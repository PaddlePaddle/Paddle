// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"

#include "stdio.h"
int main() {
  int count = 0;
  if (hipSuccess != hipGetDeviceCount(&count)) return -1;
  if (count == 0) return -1;
  for (int device = 0; device < count; ++device) {
    hipDeviceProp_t prop;
    if (hipSuccess == hipGetDeviceProperties(&prop, device))
      printf("%d.%d ", prop.major, prop.minor);
  }
  return 0;
}
