#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/op_registry.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void KeRelu3(const T* x, const int num, T* y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    y[i] = max(x[i], static_cast<T>(0.));
  }
}

template <typename DeviceContext, typename T>
class Relu3CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in_t = ctx.Input<Tensor>("X");
    auto* out_t = ctx.Output<Tensor>("Y");
    auto x = in_t->data<T>();
    auto y = out_t->mutable_data<T>(ctx.GetPlace());

    auto& dev_ctx = ctx.template device_context<DeviceContext>();

    int num = in_t->numel();
    int block = 512;
    int grid = (num + block - 1) / block;
    KeRelu3<T><<<grid, block, 0, dev_ctx.stream()>>>(x, num, y);
  }
};

template <typename T>
__global__ void KeRelu3Grad(const T* y, const T* dy, const int num, T* dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    dx[i] = dy[i] * (y[i] > 0 ? 1. : 0.);
  }
}

template <typename DeviceContext, typename T>
class Relu3GradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* dy_t = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto* y_t = ctx.Input<Tensor>("Y");
    auto* dx_t = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto dy = dy_t->data<T>();
    auto y = y_t->data<T>();
    auto dx = dx_t->mutable_data<T>(ctx.GetPlace());

    auto& dev_ctx = ctx.template device_context<DeviceContext>();

    int num = dy_t->numel();
    int block = 512;
    int grid = (num + block - 1) / block;
    KeRelu3Grad<T><<<grid, block, 0, dev_ctx.stream()>>>(y, dy, num, dx);
  }
};

}  // namespace operators
}  // namespace paddle

using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(relu3,
                        paddle::operators::Relu3CUDAKernel<CUDA, float>,
                        paddle::operators::Relu3CUDAKernel<CUDA, double>);

REGISTER_OP_CUDA_KERNEL(relu3_grad,
                        paddle::operators::Relu3GradCUDAKernel<CUDA, float>,
                        paddle::operators::Relu3GradCUDAKernel<CUDA, double>);
