#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**

This example adopts example 16 to use 3xTF32 to bring FP32 accuracy with 2x performance
compared with CUDA Cores.  See example 27 for the trick of 3xTF32. 
*/

#include <iostream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/convolution.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/reference/host/error_metrics.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

// The code section below describes datatype for input, output tensors and computation between
// elements 
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = float;                       // Data type of elements in input tensor
using ElementInputB = float;                       // Data type of elements in input tensor
using ElementOutput = float;                       // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ThreadblockShape = cutlass::gemm::GemmShape<128, 64, 16>;  // Threadblock tile shape

// This code section describes tile size a warp will compute
using WarpShape = cutlass::gemm::GemmShape<64, 32, 16>;         // Warp tile shape

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 8>;    // TensorCore instruction shape

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipelines you want to use
constexpr int NumStages = 3;

// This code section describe iterator algorithm selected is Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // The number of elements per vectorized.
                                                       // memory access. This becomes the vector width of
                                                       // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

// 3xTF32 Fprop
using Conv2dFpropKernel_3xTF32 = typename cutlass::conv::kernel::DefaultConv2dFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  // Only thing needs to be changed from normal Fprop
  cutlass::arch::OpMultiplyAddFastF32,
  IteratorAlgorithm
>::Kernel;

// 1xTF32 Fprop
using Conv2dFpropKernel_1xTF32 = typename cutlass::conv::kernel::DefaultConv2dFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  IteratorAlgorithm
>::Kernel;

using ImplicitGemm_3xTF32 = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel_3xTF32>;
using ImplicitGemm_1xTF32 = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel_1xTF32>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    benchmark(false) { }

  // Verify the problem size is compatible with the CUTLASS Convolution implementation.
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 4;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Updates input and filter sizes
  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size) {

    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c(); 

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "28_ampere_3xtf32_fast_accurate_tensorop_fprop example\n\n"
      << "  This example uses Ampere's Tensor Core operators on F16 data types to compute\n"
      << "  forward convolution on tensors of layout NHWC.\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --n=<int>            Input tensor extent N\n"
      << "  --h=<int>            Input tensor extent H\n"
      << "  --w=<int>            Input tensor extent W\n"
      << "  --c=<int>            Input tensor extent C\n"
      << "  --k=<int>            Filter extent K\n"
      << "  --r=<int>            Filter extent R\n"
      << "  --s=<int>            Filter extent S\n\n"
      << "  --alpha=<float>      Epilogue scalar alpha\n"
      << "  --beta=<float>       Epilogue scalar beta\n\n"
      << "  --benchmark          If set (true), performance benchmarking on several layers and batch-size.\n"
      << "  --iterations=<int>   Number of profiling iterations to perform.\n"
      << "  --save-workspace     If set, workspace is written to a text file.\n"
      << "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/28_ampere_3xtf32_fast_accurate_tensorop_fprop/28_ampere_3xtf32_fast_accurate_tensorop_fprop  --n=32 --h=224 --w=224 --c=128 --k=256 --r=1 --s=1\n\n"
      << "$ ./examples/28_ampere_3xtf32_fast_accurate_tensorop_fprop/28_ampere_3xtf32_fast_accurate_tensorop_fprop  --n=1 --h=224 --w=224 --c=32 --k=32 --r=3 --s=3 --ref-check\n\n";

    return out;
  }
  
  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;

  double l2_norm_3xtf32_vs_fp64;
  double l2_norm_1xtf32_vs_fp64;
  double l2_norm_fp32_vs_fp64;

  Result(): 
    runtime_ms(0), 
    gflops(0),
    status(cutlass::Status::kSuccess),
    error(hipSuccess),
    l2_norm_3xtf32_vs_fp64(0),
    l2_norm_1xtf32_vs_fp64(0),
    l2_norm_fp32_vs_fp64(0) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,Runtime,GFLOPs,3xTF32_vs_FP64,1xTF32_vs_FP64,FP32_vs_FP64";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out 
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << runtime_ms << ","
      << gflops << ","
      << l2_norm_3xtf32_vs_fp64 << ","
      << l2_norm_1xtf32_vs_fp64 << ","
      << l2_norm_fp32_vs_fp64;

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Runs one benchmark
Result profile_convolution(Options const &options) {

  Result result;

  ////////////////////////////////////////////////////////////////////////////////
  /// 1. Initialize F32 Precision input tensors using CUTLASS helper functions
  ////////////////////////////////////////////////////////////////////////////////

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  cutlass::HostTensor<float, LayoutInputA> tensor_a_F32(options.input_size);
  cutlass::HostTensor<float, LayoutInputB> tensor_b_F32(options.filter_size);
  cutlass::HostTensor<float, LayoutOutput> tensor_c_F32(options.output_size());
  cutlass::HostTensor<float, LayoutOutput> tensor_d_F32(options.output_size());

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a_F32.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8));

  // Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b_F32.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8));

  // Fill tensor C on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c_F32.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8));

  // Fill tensor D on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_d_F32.host_view());

  // Copy data from host to GPU
  tensor_a_F32.sync_device();
  tensor_b_F32.sync_device();
  tensor_c_F32.sync_device();
  tensor_d_F32.sync_device();

  ////////////////////////////////////////////////////////////////////////////////
  /// 2. Initialize F32 Precision input tensors using CUTLASS helper functions
  ////////////////////////////////////////////////////////////////////////////////

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  cutlass::HostTensor<double, LayoutInputA> tensor_a_F64(options.input_size);
  cutlass::HostTensor<double, LayoutInputB> tensor_b_F64(options.filter_size);
  cutlass::HostTensor<double, LayoutOutput> tensor_c_F64(options.output_size());

  cutlass::HostTensor<double, LayoutOutput> tensor_d_F64(options.output_size());
  cutlass::HostTensor<float, LayoutOutput> tensor_d_3xTF32(options.output_size());
  cutlass::HostTensor<float, LayoutOutput> tensor_d_1xTF32(options.output_size());

  // Copy values from the DP tensors
  cutlass::reference::host::TensorCopy(tensor_a_F64.host_view(), tensor_a_F32.host_view());
  cutlass::reference::host::TensorCopy(tensor_b_F64.host_view(), tensor_b_F32.host_view());
  cutlass::reference::host::TensorCopy(tensor_c_F64.host_view(), tensor_c_F32.host_view());
  cutlass::reference::host::TensorCopy(tensor_d_F64.host_view(), tensor_d_F32.host_view());
  cutlass::reference::host::TensorCopy(tensor_d_3xTF32.host_view(), tensor_d_F32.host_view());
  cutlass::reference::host::TensorCopy(tensor_d_1xTF32.host_view(), tensor_d_F32.host_view());
 
  // Copy data from host to GPU
  tensor_a_F64.sync_device();
  tensor_b_F64.sync_device();
  tensor_c_F64.sync_device();
  tensor_d_F64.sync_device();
  tensor_d_3xTF32.sync_device();
  tensor_d_1xTF32.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(      
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices
  );

  ////////////////////////////////////////////////////////////////////////////////
  /// 3. Run  3xTF32 kernel within a profiling loop
  ////////////////////////////////////////////////////////////////////////////////

  // Construct ImplicitGemm::Argument structure with conv2d 
  // problem size, data pointers, and epilogue values
  typename ImplicitGemm_3xTF32::Arguments arguments_3xTF32{
    problem_size,
    tensor_a_F32.device_ref(),
    tensor_b_F32.device_ref(),
    tensor_c_F32.device_ref(),
    tensor_d_3xTF32.device_ref(),
    {options.alpha, options.beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemm_3xTF32 implicit_gemm_op_3xTF32;

  size_t workspace_size_3xTF32 = implicit_gemm_op_3xTF32.get_workspace_size(arguments_3xTF32);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace_3xTF32(workspace_size_3xTF32);

  result.status = implicit_gemm_op_3xTF32.can_implement(arguments_3xTF32);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op_3xTF32.initialize(arguments_3xTF32, workspace_3xTF32.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_op_3xTF32();

  CUTLASS_CHECK(result.status);
  
  //
  // Performance measurement
  //

  hipEvent_t events[2];
  
  for (auto & event : events) {
    result.error = hipEventCreate(&event);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }
  }

  // Record an event at the start of a series of convolution operations.
  result.error = hipEventRecord(events[0]);
  if (result.error != hipSuccess) {
    std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
    return result;
  }

  // Launch a sequence of implicit GEMM operations on the device
  for (int iteration = 0; iteration < options.iterations; ++iteration) {
    result.status = implicit_gemm_op_3xTF32();
    CUTLASS_CHECK(result.status);
  }

  // Record an event when the convolutions have been launched.
  result.error = hipEventRecord(events[1]);
  if (result.error != hipSuccess) {
    std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
    return result;
  }

  // Wait for work on the device to complete.
  result.error = hipEventSynchronize(events[1]);
  if (result.error != hipSuccess) {
    std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
    return result;
  }

  // Measure elapsed runtime
  float runtime_ms = 0;
  result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  if (result.error != hipSuccess) {
    std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
    return result;
  }

  // Print average runtime and GFLOPs.
  result.runtime_ms = double(runtime_ms) / double(options.iterations);
  result.gflops = options.gflops(result.runtime_ms / 1000.0);

  // Cleanup
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }

  tensor_d_3xTF32.sync_host();

  ////////////////////////////////////////////////////////////////////////////////
  /// 4. Run 1xTF32 kernel within a profiling loop
  ////////////////////////////////////////////////////////////////////////////////

  // Construct ImplicitGemm::Argument structure with conv2d 
  // problem size, data pointers, and epilogue values
  typename ImplicitGemm_1xTF32::Arguments arguments_1xTF32{
    problem_size,
    tensor_a_F32.device_ref(),
    tensor_b_F32.device_ref(),
    tensor_c_F32.device_ref(),
    tensor_d_1xTF32.device_ref(),
    {options.alpha, options.beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemm_1xTF32 implicit_gemm_op_1xTF32;

  size_t workspace_size_1xTF32 = implicit_gemm_op_1xTF32.get_workspace_size(arguments_1xTF32);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace_1xTF32(workspace_size_1xTF32);

  result.status = implicit_gemm_op_1xTF32.can_implement(arguments_1xTF32);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op_1xTF32.initialize(arguments_1xTF32, workspace_1xTF32.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_op_1xTF32();

  CUTLASS_CHECK(result.status);
  
  tensor_d_1xTF32.sync_host();

  ////////////////////////////////////////////////////////////////////////////////
  // Run reference kernel (F64)
  ////////////////////////////////////////////////////////////////////////////////

  cutlass::reference::device::Conv2d<
    double,
    LayoutInputA,
    double,
    LayoutInputB,
    double,
    LayoutOutput,
    double,
    double 
  >(
      cutlass::conv::Operator::kFprop,
      problem_size,
      tensor_a_F64.device_ref(),
      tensor_b_F64.device_ref(),
      tensor_c_F64.device_ref(),
      tensor_d_F64.device_ref(),
      options.alpha, 
      options.beta);

  // Wait for kernels to finish
  hipDeviceSynchronize();

  // Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d_F64.sync_host();

  ////////////////////////////////////////////////////////////////////////////////
  // Run reference kernel (F32)
  ////////////////////////////////////////////////////////////////////////////////

  cutlass::reference::device::Conv2d<
    float,
    LayoutInputA,
    float,
    LayoutInputB,
    float,
    LayoutOutput,
    float,
    float 
  >(
      cutlass::conv::Operator::kFprop,
      problem_size,
      tensor_a_F32.device_ref(),
      tensor_b_F32.device_ref(),
      tensor_c_F32.device_ref(),
      tensor_d_F32.device_ref(),
      options.alpha, 
      options.beta);

  // Wait for kernels to finish
  hipDeviceSynchronize();

  // Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d_F32.sync_host();

  ////////////////////////////////////////////////////////////////////////////////
  ///////               Compute l2 norms 
  ////////////////////////////////////////////////////////////////////////////////

  // l2 norm 3xTF32 vs F64
  cutlass::HostTensor<double, LayoutOutput> tensor_d_3xTF32_in_F64(options.output_size());
  cutlass::reference::host::TensorCopy(tensor_d_3xTF32_in_F64.host_view(), tensor_d_3xTF32.host_view());

  result.l2_norm_3xtf32_vs_fp64 = cutlass::reference::host::TensorRelativeErrorMetric(
    tensor_d_3xTF32_in_F64.host_view(), tensor_d_F64.host_view());

  // l2 norm 1xTF32 vs F64
  cutlass::HostTensor<double, LayoutOutput> tensor_d_1xTF32_in_F64(options.output_size());
  cutlass::reference::host::TensorCopy(tensor_d_1xTF32_in_F64.host_view(), tensor_d_1xTF32.host_view());

  result.l2_norm_1xtf32_vs_fp64 = cutlass::reference::host::TensorRelativeErrorMetric(
    tensor_d_1xTF32_in_F64.host_view(), tensor_d_F64.host_view());

  // l2 norm F32 vs F64
  cutlass::HostTensor<double, LayoutOutput> tensor_d_F32_in_F64(options.output_size());
  cutlass::reference::host::TensorCopy(tensor_d_F32_in_F64.host_view(), tensor_d_F32.host_view());

  result.l2_norm_fp32_vs_fp64 = cutlass::reference::host::TensorRelativeErrorMetric(
    tensor_d_F32_in_F64.host_view(), tensor_d_F64.host_view());

  ///////////////////////////////////////////////////////////////////////////////

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "28_ampere_3xtf32_fast_accurate_tensorop_fprop_"
      << options.input_size.n() << "x" << options.input_size.h() << "x" << options.input_size.w() << "x" << options.input_size.c() 
      << "_"
      << options.filter_size.n() << "x" << options.filter_size.h() << "x" << options.filter_size.w() << "x" << options.filter_size.c() 
      << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace 
      << "Input = \n" << tensor_a_F32.host_view() << "\n\n"
      << "Filters = \n" << tensor_b_F32.host_view() << "\n\n";

    output_workspace << "TF32x3 = \n" << tensor_d_3xTF32.host_view() << std::endl;
    output_workspace << "TF32x1 = \n" << tensor_d_1xTF32.host_view() << std::endl;
    output_workspace << "FP32 = \n" << tensor_d_F32.host_view() << std::endl;
    output_workspace << "FP64 = \n" << tensor_d_F64.host_view() << "\n\n";

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major > 8 || (props.major == 8 && props.minor >= 0))) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // Benchmark several layers

    int batch_sizes[] = {1, 32, 64, 128, 256};

    struct Benchmark {
      int h, w, c, k, r, s;
    } layers[] = {
      {56,  56,   64,   256, 1, 1},
      {56,  56,   64,    64, 1, 1},
      {56,  56,   64,    64, 3, 3},
      {56,  56,  256,    64, 1, 1},
      {56,  56,  256,   512, 1, 1},
      {56,  56,  256,   128, 1, 1},
      {28,  28,  128,   128, 3, 3},
      {28,  28,  128,   512, 1, 1},
      {28,  28,  512,   128, 1, 1},
      {28,  28,  512,  1024, 1, 1},
      {28,  28,  512,   256, 1, 1},
      {14,  14,  256,   256, 3, 3},
      {14,  14,  256,  1024, 1, 1},
      {14,  14,  1024,  256, 1, 1},
      {14,  14,  1024, 2048, 1, 1},
      {14,  14,  1024,  512, 1, 1},
      {7,    7,   512,  512, 3, 3},
    };

    Result::print_header(std::cout, options) << std::endl;

    int idx = 1;

    for (auto const &layer : layers) {
      for (auto N : batch_sizes) {

        options.update({N, layer.h, layer.w, layer.c}, {layer.k, layer.r, layer.s, layer.c});

        Result result = profile_convolution(options);
        result.print(std::cout, idx, options) << std::endl;
      }

      ++idx;
    }
  }
  else {

    // Execute one problem size
    if (!options.valid()) {
      std::cerr << "Invalid problem." << std::endl;
      return -1;
    }

    Result result = profile_convolution(options);

    Result::print_header(std::cout, options) << std::endl;
    result.print(std::cout, 1, options) << std::endl;
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
