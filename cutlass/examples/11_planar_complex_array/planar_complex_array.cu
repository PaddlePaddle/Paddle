/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Planar Complex Array Example

  This example demonstrates the CUTLASS Library's exposure of planar complex GEMM kernels which
  execute a batch of matrix products, loading problem sizes and matrix base pointers from arrays
  in global memory.

  These kernels represent complex matrices by storing the real and imaginary parts of the matrix in
  disjoint regions in memory. These real-valued matrices are stored using existing cuBLAS layouts
  as either column-major or row-major layouts with a single leading dimension indicating the stride
  between columns or rows.

  The CUTLASS Library collects multiple template instantiations in a data structure and offers
  a BLAS-like dispatch API to invoke the appropriate kernel on the Volta or Turing architectures.

  CUTLASS decouples matrix layout from complex transformation, so four possible transformations
  are possible on the A and B operands:

    n:  column-major
    c:  column-major complex conjugate
    t:  row-major
    h:  row-major complex conjugate

  To build strictly the planar complex kernels needed for general application, execute the following
  CMake command in an empty build directory.

    $ cmake .. -DCUTLASS_NVCC_ARCHS="70;75;80" \
      -DCUTLASS_LIBRARY_KERNELS=cutlass_tensorop_*gemm_planar_complex

  This builds all planar complex GEMM variants for Volta and Turing architectures.

  To build strictly the kernels needed for this example, an even narrower filter string may be
  specified as follows. This only builds planar complex GEMMs targeting Tensor Cores for
  the 'CN' layout configuration (conjugate A operand with both A and B as column-major).

    $ cmake .. -DCUTLASS_NVCC_ARCHS="70;75;80" \
      -DCUTLASS_LIBRARY_KERNELS=cutlass_tensorop_f16_s*gemm_planar_complex_array_f16*cn

    $ make 11_planar_complex_array

    $ ./examples/11_planar_complex_array/11_planar_complex_array --m=2048 --n=1024 --k=512 --batch=10
*/

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor_planar_complex.h"

#include "cutlass/util/reference/device/tensor_fill.h"

#include "cutlass/util/reference/device/gemm_planar_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"

#include "cutlass/library/handle.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  cutlass::gemm::GemmCoord problem_size;
  int batch_count;
  cutlass::complex<float> alpha;
  cutlass::complex<float> beta;

  bool reference_check;
  int iterations;
  
  Options():
    help(false),
    problem_size({1024, 1024, 1024}),
    batch_count(1),
    reference_check(true),
    iterations(20),
    alpha(1),
    beta() { }

  bool valid() {
    return true;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("batch", batch_count);

    cmd.get_cmd_line_argument("alpha", alpha.real());
    cmd.get_cmd_line_argument("alpha_i", alpha.imag());
    cmd.get_cmd_line_argument("beta", beta.real());
    cmd.get_cmd_line_argument("beta_i", beta.imag());
    
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "11_planar_complex_array example\n\n"
      << "  This example uses the CUTLASS Library to execute Planar Complex Array GEMM computations.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --batch=<int>               Number of GEMM operations executed in one batch\n"
      << "  --alpha=<f32>               Epilogue scalar alpha (real part)\n"
      << "  --alpha_i=<f32>             Epilogue scalar alpha (imaginary part)\n"
      << "  --beta=<f32>                Epilogue scalar beta (real part)\n\n"
      << "  --beta_i=<f32>              Epilogue scalar beta (imaginary part)\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/11_planar_complex_array/11_planar_complex_array\n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = problem_size.product() * batch_count * 4;
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Performance test environment for planar complex
class TestbedPlanarComplex {
public:

  // Half-precision input and output
  using Element = cutlass::half_t;

  // Configurations for layouts and internal computation
  using LayoutA = cutlass::layout::ColumnMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementCompute = float;
  using ElementAccumulator = float;

  //
  // Data members
  //

  cutlass::library::Handle handle;

  cutlass::gemm::GemmCoord problem_size;
  int batch_count;
  cutlass::DeviceAllocation<Element> tensor_A;
  cutlass::DeviceAllocation<Element> tensor_B;
  cutlass::DeviceAllocation<Element> tensor_C;
  cutlass::DeviceAllocation<Element> tensor_D;
  cutlass::DeviceAllocation<Element> tensor_D_ref;

  cutlass::DeviceAllocation<void *> ptr_A_real;
  cutlass::DeviceAllocation<void *> ptr_A_imag;
  cutlass::DeviceAllocation<void *> ptr_B_real;
  cutlass::DeviceAllocation<void *> ptr_B_imag;
  cutlass::DeviceAllocation<void *> ptr_C_real;
  cutlass::DeviceAllocation<void *> ptr_C_imag;
  cutlass::DeviceAllocation<void *> ptr_D_real;
  cutlass::DeviceAllocation<void *> ptr_D_imag;

  //
  // Methods
  //

  TestbedPlanarComplex(
    Options const &options
  ): 
    problem_size(options.problem_size), batch_count(options.batch_count) {

    // Allocate device memory for batched planar complex GEMM
    tensor_A.reset(int64_t(problem_size.m()) * problem_size.k() * batch_count * 2);
    tensor_B.reset(int64_t(problem_size.k()) * problem_size.n() * batch_count * 2);
    tensor_C.reset(int64_t(problem_size.m()) * problem_size.n() * batch_count * 2);
    tensor_D.reset(int64_t(problem_size.m()) * problem_size.n() * batch_count * 2);
    tensor_D_ref.reset(int64_t(problem_size.m()) * problem_size.n() * batch_count * 2);

    ptr_A_real.reset(batch_count);
    ptr_A_imag.reset(batch_count);
    ptr_B_real.reset(batch_count);
    ptr_B_imag.reset(batch_count);
    ptr_C_real.reset(batch_count);
    ptr_C_imag.reset(batch_count);
    ptr_D_real.reset(batch_count);
    ptr_D_imag.reset(batch_count);

  }

  void initialize() {

    uint64_t seed = 1073;

    // Use small integers to simplify correctness checking
    int scope_max = 6;
    int scope_min = -6;

    cutlass::reference::device::BlockFillRandomUniform(
        tensor_A.get(), tensor_A.size(), seed, Element(scope_max), Element(scope_min), 0);

    cutlass::reference::device::BlockFillRandomUniform(
        tensor_B.get(), tensor_B.size(), seed * 2019, Element(scope_max), Element(scope_min), 0);

    cutlass::reference::device::BlockFillRandomUniform(
        tensor_C.get(), tensor_C.size(), seed * 2020, Element(scope_max), Element(scope_min), 0);
  }

  Result profile(Options const &options) {

    Result result;

    initialize();

    Element *ptr_A = tensor_A.get();
    Element *ptr_B = tensor_B.get();
    Element *ptr_C = tensor_C.get();
    Element *ptr_D = tensor_D.get();

    int64_t batch_stride_A = int64_t(problem_size.m()) * problem_size.k() * 2;
    int64_t batch_stride_B = int64_t(problem_size.k()) * problem_size.n() * 2;
    int64_t batch_stride_C = int64_t(problem_size.m()) * problem_size.n() * 2;
    int64_t batch_stride_D = int64_t(problem_size.m()) * problem_size.n() * 2;

    typename LayoutA::Stride::Index lda = LayoutA::packed({problem_size.m(), problem_size.k()}).stride(0);
    typename LayoutB::Stride::Index ldb = LayoutB::packed({problem_size.k(), problem_size.n()}).stride(0);
    typename LayoutC::Stride::Index ldc = LayoutC::packed({problem_size.m(), problem_size.n()}).stride(0);
    typename LayoutC::Stride::Index ldd = LayoutC::packed({problem_size.m(), problem_size.n()}).stride(0);


    int64_t imag_stride_A = int64_t(problem_size.m()) * problem_size.k();
    int64_t imag_stride_B = int64_t(problem_size.k()) * problem_size.n();
    int64_t imag_stride_C = int64_t(problem_size.m()) * problem_size.n();
    int64_t imag_stride_D = int64_t(problem_size.m()) * problem_size.n();
    
    //
    // Configure pointers in global memory
    //

    struct {
      Element *base;
      void **ptr_real;
      void **ptr_imag;
      int64_t batch_stride;
      int64_t imag_stride;
    } tensors[] = {
      { tensor_A.get(), ptr_A_real.get(), ptr_A_imag.get(), batch_stride_A, imag_stride_A},
      { tensor_B.get(), ptr_B_real.get(), ptr_B_imag.get(), batch_stride_B, imag_stride_B},
      { tensor_C.get(), ptr_C_real.get(), ptr_C_imag.get(), batch_stride_C, imag_stride_C},
      { tensor_D.get(), ptr_D_real.get(), ptr_D_imag.get(), batch_stride_D, imag_stride_D}
    };

    for (auto const &tensor : tensors) {
      for (int idx = 0; idx < batch_count; ++idx) {

        void *ptr_real = tensor.base + idx * tensor.batch_stride;
        void *ptr_imag = tensor.base + idx * tensor.batch_stride + tensor.imag_stride;

        hipError_t error = hipMemcpy(
          tensor.ptr_real + idx,
          &ptr_real,
          sizeof(void *),
          hipMemcpyHostToDevice);

        if (error != hipSuccess) {
          throw std::runtime_error("Failed to copy pointer to device memory");
        }

        error = hipMemcpy(
          tensor.ptr_imag + idx,
          &ptr_imag,
          sizeof(void *),
          hipMemcpyHostToDevice);

        if (error != hipSuccess) {
          throw std::runtime_error("Failed to copy pointer to device memory");
        }
      }
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMM operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < options.iterations; ++iter) {

      //
      // Execute the planar complex array GEMM kernel via the CUTLASS Library's
      // dispatch routines.
      //
      // Note, for planar complex array GEMM kernels, all numeric type arguments 
      // specify the data type of the base real types. These are understood to
      // apply to planar complex representations of matrices in memory and to complex<T>
      // structures for scalars.
      //
      // See tools/library/include/cutlass/library/handle.h for more details.
      //

      result.status = handle.gemm_planar_complex_array(

        problem_size.m(),                                 // expected GEMM M dimension
        problem_size.n(),                                 // expected GEMM N dimension
        problem_size.k(),                                 // expected GEMM K dimension
        batch_count,                                      // Number of batched elements

        nullptr,
        nullptr,
        nullptr,

        cutlass::library::NumericTypeID::kF32,            // Base data type of complex-valued accumulation
        cutlass::library::NumericTypeID::kF32,            // Base data type of complex-valued alpha/beta scalars

        &options.alpha,                                   // Pointer to alpha scalar, of type complex<T>

        cutlass::library::NumericTypeID::kF16,            // Base data type of complex-valued A matrix
        cutlass::library::LayoutTypeID::kColumnMajor,     // Layout of A matrix
        cutlass::library::ComplexTransform::kConjugate,   // Complex transformation on A matrix operand

        ptr_A_real.get(),                                 // Pointer to array of pointers to real part of A matrix
        ptr_A_imag.get(),                                 // Pointer to array of pointers to imaginary part of A matrix

        lda,                                              // Leading dimension of real part of A matrix
        lda,                                              // Leading dimension of imaginary part of A matrix

        cutlass::library::NumericTypeID::kF16,            // Base data type of complex-valued B matrix
        cutlass::library::LayoutTypeID::kColumnMajor,     // Layout of B matrix
        cutlass::library::ComplexTransform::kNone,        // Complex transformation on B matrix operand

        ptr_B_real.get(),                                 // Pointer to array of pointers to real part of B matrix
        ptr_B_imag.get(),                                 // Pointer to array of pointers to imaginary part of B matrix

        ldb,                                              // Leading dimension of real part of B matrix
        ldb,                                              // Leading dimension of imaginary part of B matrix

        &options.beta,                                    // Pointer to beta scalar, of type complex<T>

        cutlass::library::NumericTypeID::kF16,            // Base data type of complex valued C and D matrices

        ptr_C_real.get(),                                 // Pointer to array of pointers to real part of C matrix
        ptr_C_imag.get(),                                 // Pointer to array of pointers to imaginary part of C matrix

        ldc,                                              // Leading dimension of real part of C matrix
        ldc,                                              // Leading dimension of imaginary part of C matrix

        ptr_D_real.get(),                                 // Pointer to array of pointers to real part of D matrix
        ptr_D_imag.get(),                                 // Pointer to array of pointers to imaginary part of D matrix

        ldd,                                              // Leading dimension of real part of D matrix
        ldd                                               // Leading dimension of imaginary part of D matrix
      );

      if (result.status != cutlass::Status::kSuccess) {
        std::cerr << "CUTLASS internal error - configuration not supported" << std::endl;
        return result;
      }
    }
    
    //
    // Stop profiling loop
    //

    // Record an event when the GEMM operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    if (handle.get_last_operation()) {
      std::cout << "Recently executed '" << handle.get_last_operation()->description().name << "'" << std::endl;
    }

    //
    // Compute reference in device code
    //

    if (options.reference_check) {

      result.passed = true;

      for (int64_t idx = 0; result.passed && idx < int64_t(batch_count); ++idx) {
        cutlass::reference::device::GemmPlanarComplex<
          Element, LayoutA,
          Element, LayoutB,
          Element, LayoutC,
          ElementAccumulator
        >(
          problem_size,
          options.alpha,
          {tensor_A.get() + idx * batch_stride_A, lda, imag_stride_A},
          cutlass::ComplexTransform::kConjugate,
          {tensor_B.get() + idx * batch_stride_B, ldb, imag_stride_B},
          cutlass::ComplexTransform::kNone,
          options.beta,
          {tensor_C.get() + idx * batch_stride_C, ldc, imag_stride_C},
          {tensor_D_ref.get() + idx * batch_stride_D, ldd, imag_stride_D}
        );

        Element epsilon = 0.1_hf;
        Element nonzero_floor = 0.1_hf;

        result.passed = cutlass::reference::device::BlockCompareRelativelyEqual(
          tensor_D.get() + idx * batch_stride_D,
          tensor_D_ref.get() + idx * batch_stride_D,
          batch_stride_D,
          epsilon,
          nonzero_floor
        );
      }

      if (result.passed) {
        std::cout << "Reference check passed." << std::endl;
      }
      else {
        std::cerr << "Error - reference check failed." << std::endl;
      }
    }

    std::cout << "Runtime: " << result.runtime_ms << " ms" << std::endl;
    std::cout << " GFLOPs: " << result.gflops << std::endl;

    return result;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //
  // Volta Tensor Core operations are first available in CUDA 10.1 Toolkit.
  //
  // Turing Tensor Core operations are first available in CUDA 10.2 Toolkit.
  //

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major < 7) {
    std::cerr << "Tensor Core operations must be run on a machine with compute capability at least 70."
              << std::endl;

    // Returning zero so this passes on older architectures. Its actions are no-op.
    return 0;
  }
  else if (props.major == 7 && props.minor <= 2) {
    //
    // If running on the Volta architecture, at least CUDA 10.1 Toolkit is required to run this example.
    //
    if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 1))) {
      std::cerr << "Volta Tensor Core operations must be compiled with CUDA 10.1 Toolkit or later." << std::endl;
      
      // Returning zero so this passes on older Toolkits. Its actions are no-op.
      return 0;
    }
  }
  else if (props.major == 7 && props.minor >= 5) {
    //
    // If running on the Turing architecture, at least CUDA 10.2 Toolkit is required to run this example.
    //
    if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
      std::cerr << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;
      
      // Returning zero so this passes on older Toolkits. Its actions are no-op.
      return 0;
    }
  }
  else {
    // NVIDIA Ampere Architecture GPUs (SM80 and later) are fully supported on CUDA 11 Toolkit and beyond.
    //
    // fall through
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  // Execute one problem size
  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }

  TestbedPlanarComplex testbed(options);

  Result result = testbed.profile(options);

  return result.passed ? 0 : -1;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

