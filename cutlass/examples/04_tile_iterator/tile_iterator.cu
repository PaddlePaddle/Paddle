#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to use the PredicatedTileIterator in CUTLASS to load data from
  addressable memory, and then store it back into addressable memory.

  TileIterator is a core concept in CUTLASS that enables efficient loading and storing of data to
  and from addressable memory. The PredicateTileIterator accepts a ThreadMap type, which defines
  the mapping of threads to a "tile" in memory. This separation of concerns enables user-defined
  thread mappings to be specified. 

  In this example, a PredicatedTileIterator is used to load elements from a tile in global memory,
  stored in column-major layout, into a fragment and then back into global memory in the same
  layout.

  This example uses CUTLASS utilities to ease the matrix operations.

*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>
#include <fstream>

// CUTLASS includes
#include "cutlass/transform/threadblock/predicated_tile_iterator.h"
#include "cutlass/layout/pitch_linear.h"
#include "cutlass/transform/pitch_linear_thread_map.h"

//
//  CUTLASS utility includes
//

// Defines operator<<() to write TensorView objects to std::ostream
#include "cutlass/util/tensor_view_io.h"

// Defines cutlass::HostTensor<>
#include "cutlass/util/host_tensor.h"

// Defines cutlass::reference::host::TensorFill() and
// cutlass::reference::host::TensorFillBlockSequential()
#include "cutlass/util/reference/host/tensor_fill.h"

#pragma warning( disable : 4503)
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define PredicatedTileIterators to load and store a M-by-K tile, in column major layout.

template <typename Iterator>
__global__ void copy(
    typename Iterator::Params dst_params,
    typename Iterator::Element *dst_pointer,
    typename Iterator::Params src_params,
    typename Iterator::Element *src_pointer,
    cutlass::Coord<2> extent) {


    Iterator dst_iterator(dst_params, dst_pointer, extent, threadIdx.x);
    Iterator src_iterator(src_params, src_pointer, extent, threadIdx.x);

    // PredicatedTileIterator uses PitchLinear layout and therefore takes in a PitchLinearShape.
    // The contiguous dimension can be accessed via Iterator::Shape::kContiguous and the strided
    // dimension can be accessed via Iterator::Shape::kStrided
    int iterations = (extent[1] + Iterator::Shape::kStrided - 1) / Iterator::Shape::kStrided;

    typename Iterator::Fragment fragment;

    for(int i = 0; i < fragment.size(); ++i) {
      fragment[i] = 0;
    }

    src_iterator.load(fragment);
    dst_iterator.store(fragment);


    ++src_iterator;
    ++dst_iterator;

    for(; iterations > 1; --iterations) {

      src_iterator.load(fragment);
      dst_iterator.store(fragment);

      ++src_iterator;
      ++dst_iterator;
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

// Initializes the source tile with sequentially increasing values and performs the copy into
// the destination tile using two PredicatedTileIterators, one to load the data from addressable
// memory into a fragment (regiser-backed array of elements owned by each thread) and another to 
// store the data from the fragment back into the addressable memory of the destination tile.

hipError_t TestTileIterator(int M, int K) {

    // For this example, we chose a <64, 4> tile shape. The PredicateTileIterator expects
    // PitchLinearShape and PitchLinear layout.
    using Shape = cutlass::layout::PitchLinearShape<64, 4>;
    using Layout = cutlass::layout::PitchLinear;
    using Element = int;
    int const kThreads = 32;

    // ThreadMaps define how threads are mapped to a given tile. The PitchLinearStripminedThreadMap
    // stripmines a pitch-linear tile among a given number of threads, first along the contiguous
    // dimension then along the strided dimension.
    using ThreadMap = cutlass::transform::PitchLinearStripminedThreadMap<Shape, kThreads>;

    // Define the PredicateTileIterator, using TileShape, Element, Layout, and ThreadMap types
    using Iterator = cutlass::transform::threadblock::PredicatedTileIterator<
        Shape, Element, Layout, 1, ThreadMap>;


    cutlass::Coord<2> copy_extent = cutlass::make_Coord(M, K);
    cutlass::Coord<2> alloc_extent = cutlass::make_Coord(M, K);

    // Allocate source and destination tensors
    cutlass::HostTensor<Element, Layout> src_tensor(alloc_extent);
    cutlass::HostTensor<Element, Layout> dst_tensor(alloc_extent);

    Element oob_value = Element(-1);

    // Initialize destination tensor with all -1s
    cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
    // Initialize source tensor with sequentially increasing values
    cutlass::reference::host::BlockFillSequential(src_tensor.host_data(), src_tensor.capacity());

    dst_tensor.sync_device();
    src_tensor.sync_device();

    typename Iterator::Params dst_params(dst_tensor.layout());
    typename Iterator::Params src_params(src_tensor.layout());

    dim3 block(kThreads, 1);
    dim3 grid(1, 1);

    // Launch copy kernel to perform the copy
    copy<Iterator><<< grid, block >>>(
            dst_params,
            dst_tensor.device_data(),
            src_params,
            src_tensor.device_data(),
            copy_extent
    );

    hipError_t result = hipGetLastError();
    if(result != hipSuccess) {
      std::cerr << "Error - kernel failed." << std::endl;
      return result;
    }

    dst_tensor.sync_host();

    // Verify results
    for(int s = 0; s < alloc_extent[1]; ++s) {
      for(int c = 0; c < alloc_extent[0]; ++c) {

          Element expected = Element(0);

          if(c < copy_extent[0] && s < copy_extent[1]) {
            expected = src_tensor.at({c, s});
          }
          else {
            expected = oob_value;
          }

          Element got = dst_tensor.at({c, s});
          bool equal = (expected == got);

          if(!equal) {
              std::cerr << "Error - source tile differs from destination tile." << std::endl;
            return hipErrorUnknown;
          }
      }
    }

    return hipSuccess;
}

int main(int argc, const char *arg[]) {

    hipError_t result = TestTileIterator(57, 35);

    if(result == hipSuccess) {
      std::cout << "Passed." << std::endl;  
    }

    // Exit
    return result == hipSuccess ? 0 : -1;
}

