/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include <iostream>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/device/gemm_array.h"
#include "cutlass/gemm/device/gemm_batched.h"

#pragma warning( disable : 4503)

/*
This example demonstrates how to use cutlass to compute a batched strided gemm in two different ways:
  1. By specifying pointers to the first matrices of the batch and the stride between the consecutive
     matrices of the batch (this is called a strided batched gemm).
  2. By copying pointers to all matrices of the batch to the device memory (this is called an array gemm).
In this example, both A and B matrix are non-transpose and column major matrix
batched_C = batched_A x batched_B
As an example, matrix C can be seen as
-----------------------------------------------------------
(0,0,0) | (0,0,1) | (0,0,2) | (1,0,0) | (1,0,1) | (1,0,2) |
-----------------------------------------------------------
(0,1,0) | (0,1,1) | (0,1,2) | (1,1,0) | (1,1,1) | (1,1,2) |
-----------------------------------------------------------
(0,2,0) | (0,2,1) | (0,2,2) | (1,2,0) | (1,2,1) | (1,2,2) |
-----------------------------------------------------------
(0,3,0) | (0,3,1) | (0,3,2) | (1,3,0) | (1,3,1) | (1,3,2) |
-----------------------------------------------------------
(0,4,0) | (0,4,1) | (0,4,2) | (1,4,0) | (1,4,1) | (1,4,2) |
-----------------------------------------------------------
(0,5,0) | (0,5,1) | (0,5,2) | (1,5,0) | (1,5,1) | (1,5,2) |
-----------------------------------------------------------
           batch 0          |           batch 1
where we denote each element with (batch_idx, row_idx, column_idx)
In this example, batch size is 2, M is 6 and N is 3
The stride (batch_stride_C) between the first element of two batches is ldc * n

matrix A can be seen as
---------------------------------------
(0,0,0) | (0,0,1) | (1,0,0) | (1,0,1) |
---------------------------------------
(0,1,0) | (0,1,1) | (1,1,0) | (1,1,1) |
---------------------------------------
(0,2,0) | (0,2,1) | (1,2,0) | (1,2,1) |
---------------------------------------
(0,3,0) | (0,3,1) | (1,3,0) | (1,3,1) |
---------------------------------------
(0,4,0) | (0,4,1) | (1,4,0) | (1,4,1) |
---------------------------------------
(0,5,0) | (0,5,1) | (1,5,0) | (1,5,1) |
---------------------------------------
     batch 0      |      batch 1
, where batch size is 2, M is 6 and K is 2
The stride (batch_stride_B) between the first element of two batches is lda * k

matrix B can be seen as
-----------------------------
(0,0,0) | (0,0,1) | (0,0,2) |
----------------------------- batch 0
(0,1,0) | (0,1,1) | (0,1,2) |
-------------------------------------
(1,0,0) | (1,0,1) | (1,0,2) |
----------------------------- batch 1
(1,1,0) | (1,1,1) | (1,1,2) |
-----------------------------
, where the batch size is 2, N is 3 and K is 2
The stride (batch_stride_C) between the first element of two batches is k


*/

hipError_t cutlass_array_sgemm(
  int m,
  int n,
  int k,
  float alpha,
  float const * const *A,
  int lda,
  float const * const *B,
  int ldb,
  float * const *C,
  int ldc,
  float beta,
  int batch_count) {

  using Gemm = cutlass::gemm::device::GemmArray<
    float, cutlass::layout::ColumnMajor,
    float, cutlass::layout::ColumnMajor,
    float, cutlass::layout::ColumnMajor
  >;

  Gemm gemm_op;

  cutlass::Status status = gemm_op({
    {m, n, k},
    A, lda,
    B, ldb,
    C, ldc,
    C, ldc,
    {alpha, beta},
    batch_count
  });

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  return hipSuccess;
}

hipError_t cutlass_strided_batched_sgemm(
  int m, 
  int n,
  int k,
  float alpha,
  float const *A,
  int lda,
  long long int batch_stride_A,
  float const *B,
  int ldb,
  long long int batch_stride_B,
  float *C,
  int ldc,
  long long int batch_stride_C,
  float beta,
  int batch_count) {

  using Gemm = cutlass::gemm::device::GemmBatched<
    float, cutlass::layout::ColumnMajor,
    float, cutlass::layout::ColumnMajor,
    float, cutlass::layout::ColumnMajor
  >;

  Gemm gemm_op;

  cutlass::Status status = gemm_op({
    {m, n, k},
    {A, lda}, 
    batch_stride_A,
    {B, ldb}, 
    batch_stride_B,
    {C, ldc}, 
    batch_stride_C,
    {C, ldc}, 
    batch_stride_C,
    {alpha, beta},
    batch_count
  });

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  return hipSuccess;
}

template<typename T> 
hipError_t strided_batched_gemm_nn_reference(
  int m,
  int n,
  int k,
  T alpha,
  std::vector<T> const &A, 
  int lda,
  long long int batch_stride_A,
  std::vector<T> const &B, 
  int ldb,
  long long int batch_stride_B,
  std::vector<T> &C, 
  int ldc,
  long long int batch_stride_C,
  T beta,
  int batch_count) {
  /*
  strided batched gemm NN
  */
  
  hipError_t result = hipSuccess;

  if (A.size() < lda * k * batch_count) {
    std::cout << "the size of A is too small" << std::endl;
    return hipErrorInvalidValue;
  }
  if (B.size() < ldb * n) {
    std::cout << "the size of B is too small" << std::endl;
    return hipErrorInvalidValue;
  }
  if (C.size() < ldc * n * batch_count) {
    std::cout << "the size of C is too small" << std::endl;
    return hipErrorInvalidValue;
  }
  
  for (int batch_idx = 0; batch_idx < batch_count; batch_idx++) {
    for (int n_idx = 0; n_idx < n; n_idx++) {
      for (int m_idx = 0; m_idx < m; m_idx++) {
        T accum = beta * C[batch_idx * batch_stride_C + n_idx * ldc + m_idx];
        for (int k_idx = 0; k_idx < k; k_idx++) {
          accum += alpha 
            * A[batch_idx * batch_stride_A + k_idx * lda + m_idx]
            * B[batch_idx * batch_stride_B + n_idx * ldb + k_idx];
        }
        C[batch_idx * batch_stride_C + n_idx * ldc + m_idx] = accum;
      }
    }
  }

  return result;
}


hipError_t run_batched_gemm(bool use_array) {

  const char* gemm_desc = use_array ? "array" : "strided batched";
  std::cout << "Running " << gemm_desc << " gemm" << std::endl;

  // Arbitrary problem size
  int const m = 520;
  int const n = 219;
  int const k = 129;
  int const batch_count = 17;

  // A, B are non-transpose, column major
  int const lda = m;
  int const ldb = k * batch_count;
  int const ldc = m;

  int const count_A = batch_count * lda * k;
  int const count_B = ldb * n;
  int const count_C = batch_count * ldc * n;

  // the memory is batched along K dimension
  long long int batch_stride_A = static_cast<long long int>(lda) * static_cast<long long int>(k);
  long long int batch_stride_B = static_cast<long long int>(k);
  long long int batch_stride_C = static_cast<long long int>(ldc) * static_cast<long long int>(n);

  // alpha and beta
  float alpha = 1.0f;
  float beta = 2.0f;

  hipError_t result = hipSuccess;

  // allocate the host memory
  std::vector<float> host_A(count_A);
  std::vector<float> host_B(count_B);
  std::vector<float> host_C(count_C);
  std::vector<float> result_C(count_C);

  // allocate the device memory
  float *A;
  float *B;
  float *C;

  result = hipMalloc(&A, count_A * sizeof(float));
  if (result != hipSuccess) {
    std::cerr << "hipMalloc result = " << result << std::endl;
    return result;
  }
  result = hipMalloc(&B, count_B * sizeof(float));
  if (result != hipSuccess) {
    std::cerr << "hipMalloc result = " << result << std::endl;
    return result;
  }
  result = hipMalloc(&C, count_C * sizeof(float));
  if (result != hipSuccess) {
    std::cerr << "hipMalloc result = " << result << std::endl;
    return result;
  }

  // Limit range to avoid floating-point errors
  int const kRange = 8;

  // fill A
  for (int b_idx = 0; b_idx < batch_count; b_idx++) {
    for (int col_idx = 0; col_idx < k; col_idx++) {
      for (int row_idx = 0; row_idx < m; row_idx++) {
        host_A[row_idx + col_idx * lda + b_idx * lda * k] = static_cast<float>((row_idx + col_idx * lda + b_idx * lda * k) % kRange);
      }
    }
  }
  // fill B
  for (int b_idx = 0; b_idx < batch_count; b_idx++) {
    for (int col_idx = 0; col_idx < n; col_idx++) {
      for (int row_idx = 0; row_idx < k; row_idx++) {
        host_B[row_idx + col_idx * ldb + b_idx * k] = static_cast<float>(((n + k * ldb + batch_count * k) - (row_idx + col_idx * ldb + b_idx * k)) % kRange);
      }
    }
  }
  // fill C
  for (int b_idx = 0; b_idx < batch_count; b_idx++) {
    for (int col_idx = 0; col_idx < n; col_idx++) {
      for (int row_idx = 0; row_idx < m; row_idx++) {
        host_C[row_idx + col_idx * ldc + b_idx * ldc * n] = 1.f;
      }
    }
  }

  // ref memory
  std::vector<float> ref_A(host_A);
  std::vector<float> ref_B(host_B);
  std::vector<float> ref_C(host_C);
  // copy host memory to device
  result = hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }
  result = hipMemcpy(B, host_B.data(), count_B * sizeof(float), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }
  result = hipMemcpy(C, host_C.data(), count_C * sizeof(float), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }

  // run cutlass
  if (use_array) {
    // allocate the host memory for the pointers to the matrices of the batch
    std::vector<float*> host_ptr_A(batch_count);
    std::vector<float*> host_ptr_B(batch_count);
    std::vector<float*> host_ptr_C(batch_count);

    // permute the batch elements to emphasize that GemmArray does not depend on matrices being separated by a fixed stride
    std::vector<size_t> permutation = {14, 11, 3, 10, 1, 13, 9, 4, 6, 16, 8, 15, 7, 12, 0, 2, 5};
    for (size_t b_idx = 0; b_idx < batch_count; b_idx++) {
      host_ptr_A[b_idx] = A + permutation[b_idx] * batch_stride_A;
      host_ptr_B[b_idx] = B + permutation[b_idx] * batch_stride_B;
      host_ptr_C[b_idx] = C + permutation[b_idx] * batch_stride_C;
    }

    // allocate the corresponding device memory
    float const **ptr_A;
    float const **ptr_B;
    float **ptr_C;

    result = hipMalloc(&ptr_A, batch_count * sizeof(float*));
    if (result != hipSuccess) {
      std::cerr << "hipMalloc result = " << result << std::endl;
      return result;
    }
    result = hipMalloc(&ptr_B, batch_count * sizeof(float*));
    if (result != hipSuccess) {
      std::cerr << "hipMalloc result = " << result << std::endl;
      return result;
    }
    result = hipMalloc(&ptr_C, batch_count * sizeof(float*));
    if (result != hipSuccess) {
      std::cerr << "hipMalloc result = " << result << std::endl;
      return result;
    }

    // copy the matrix pointers to the device
    result = hipMemcpy(ptr_A, host_ptr_A.data(), batch_count * sizeof(float*), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
      std::cerr << "hipMemcpy result = " << result << std::endl;
      return result;
    }
    result = hipMemcpy(ptr_B, host_ptr_B.data(), batch_count * sizeof(float*), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
      std::cerr << "hipMemcpy result = " << result << std::endl;
      return result;
    }
    result = hipMemcpy(ptr_C, host_ptr_C.data(), batch_count * sizeof(float*), hipMemcpyHostToDevice);
    if (result != hipSuccess) {
      std::cerr << "hipMemcpy result = " << result << std::endl;
      return result;
    }

    result = cutlass_array_sgemm(m, n, k, alpha, ptr_A, lda, ptr_B, ldb, ptr_C, ldc, beta, batch_count);

    if (result != hipSuccess)
      return result;
  } else {
    result = cutlass_strided_batched_sgemm(
      m, n, k, alpha, A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C,
      beta, batch_count);
    if (result != hipSuccess)
      return result;
  }

  // copy device memory to host
  result = hipMemcpy(result_C.data(), C, count_C * sizeof(float), hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }

  //compare with reference code
  result = strided_batched_gemm_nn_reference(m, n, k, alpha, ref_A, lda, batch_stride_A, ref_B, ldb, batch_stride_B, ref_C, ldc, batch_stride_C,
    beta, batch_count);
  if (result != 0)
    return result;

  // Expect bit-level accuracy for this simple example
  if (ref_C != result_C) {
    std::cout << "CUTLASS " << gemm_desc << " gemm does not run correctly" << std::endl;
    return hipErrorUnknown;
  }

  // free memory
  result = hipFree(A);
  if (result != hipSuccess) {
    std::cerr << "hipFree result = " << result << std::endl;
    return result;
  }
  result = hipFree(B);
  if (result != hipSuccess) {
    std::cerr << "hipFree result = " << result << std::endl;
    return result;
  }
  result = hipFree(C);
  if (result != hipSuccess) {
    std::cerr << "hipFree result = " << result << std::endl;
    return result;
  }

  return result;
}

int main() {

  hipError_t result = hipSuccess;
  for (bool use_array : {false, true}) {
    result = run_batched_gemm(use_array);
    if (result == hipSuccess) {
      std::cout << "Passed." << std::endl;
    } else {
      break;
    }
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}
