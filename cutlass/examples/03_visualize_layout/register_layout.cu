#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
  \brief CUTLASS layout visualization example
*/

#include <map>
#include <memory>

#include "cutlass/layout/matrix.h"
#include "cutlass/layout/pitch_linear.h"
#include "cutlass/layout/tensor_op_multiplicand_sm70.h"
#include "cutlass/layout/tensor_op_multiplicand_sm75.h"
#include "cutlass/layout/tensor_op_multiplicand_sm80.h"

#include "visualize_layout.h"
#include "register_layout.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

void RegisterLayouts(std::map<std::string, std::unique_ptr<VisualizeLayoutBase> > &layouts) {

  struct {
    char const *name;
    VisualizeLayoutBase *ptr;
  } layout_pairs[] = {

      {"PitchLinear", new VisualizeLayout<cutlass::layout::PitchLinear>},
      {"ColumnMajor", new VisualizeLayout<cutlass::layout::ColumnMajor>},
      {"RowMajor", new VisualizeLayout<cutlass::layout::RowMajor>},
      {"ColumnMajorInterleaved<4>",
       new VisualizeLayout<cutlass::layout::ColumnMajorInterleaved<4>>},
      {"RowMajorInterleaved<4>",
       new VisualizeLayout<cutlass::layout::RowMajorInterleaved<4>>},
      // All Ampere/Turing H/Integer matrix multiply tensor core kernels uses the same swizzling
      // layout implementation with different templates.
      //
      // BMMA 88128  Interleaved-256
      // BMMA 168256 Interleaved-256
      {"TensorOpMultiplicand<1,256>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<1, 256>>},
      // BMMA 88128  TN kblock512
      // BMMA 168256 TN kblock512
      {"TensorOpMultiplicand<1,512>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<1, 512>>},
      // BMMA 168256 TN kblock1024
      {"TensorOpMultiplicand<1,1024>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<1, 1024>>},
      // Integer matrix multiply.int4 8832  Interleaved-64
      // Integer matrix multiply.int4 16864 Interleaved-64
      {"TensorOpMultiplicand<4,64>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<4, 64>>},
      // Integer matrix multiply.int4 8832  TN kblock128
      // Integer matrix multiply.int4 16864 TN kblock128
      {"TensorOpMultiplicand<4,128>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<4, 128>>},
      // Integer matrix multiply.int4 16864 TN kblock256
      {"TensorOpMultiplicand<4,256>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<4, 256>>},
      // Integer matrix multiply 8816  Interleaved-32
      // Integer matrix multiply 16832 Interleaved-32
      {"TensorOpMultiplicand<8,32>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<8, 32>>},
      // Integer matrix multiply 8816  TN kblock64
      // Integer matrix multiply 16832 TN kblock64
      {"TensorOpMultiplicand<8,64>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<8, 64>>},
      // Integer matrix multiply 16832 TN kblock128
      {"TensorOpMultiplicand<8,128>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<8, 128>>},
      // Matrix Multiply 1688  TN kblock32
      // Matrix multiply 16816 TN kblock32
      {"TensorOpMultiplicand<16,32>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<16, 32>>},
      // Matrix multiply 1688  NT
      // Matrix multiply 16816 NT
      // Matrix multiply 16816 TN kblock64
      {"TensorOpMultiplicand<16,64>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<16, 64>>},
      // Matrix multiply 1688.TF32 TN kblock16
      {"TensorOpMultiplicand<32,16>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<32, 16>>},
      // Matrix multiply 1688.TF32 TN kblock32
      {"TensorOpMultiplicand<32,32>",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand<32, 32>>},
      // Matrix multiply 1688 NT
      {"TensorOpMultiplicandCongruous<32,32>",
       new VisualizeLayout<
           cutlass::layout::TensorOpMultiplicandCongruous<32, 32>>},
      // Matrix multiply 884 NT
      {"TensorOpMultiplicandCongruous<64,16>",
       new VisualizeLayout<
           cutlass::layout::TensorOpMultiplicandCongruous<64, 16>>},
      // Matrix multiply 884 TN
      {"TensorOpMultiplicand64bCrosswise",
       new VisualizeLayout<cutlass::layout::TensorOpMultiplicand64bCrosswise>},
      {"TensorOpMultiplicandCongruous<128,4>",
       new VisualizeLayout<
           cutlass::layout::TensorOpMultiplicandCongruous<128, 4>>},
      {"TensorOpMultiplicandCrosswise<128,4>",
       new VisualizeLayout<
           cutlass::layout::TensorOpMultiplicandCrosswise<128, 4>>},
      {"VoltaTensorOpMultiplicandCongruous<16>",
       new VisualizeLayout<
           cutlass::layout::VoltaTensorOpMultiplicandCongruous<16>>},
      {"VoltaTensorOpMultiplicandCrosswise<16,32>",
       new VisualizeLayout<
           cutlass::layout::VoltaTensorOpMultiplicandCrosswise<16, 32>>}
  };

  for (auto layout : layout_pairs) {
    layouts.emplace(std::string(layout.name), std::unique_ptr<VisualizeLayoutBase>(layout.ptr));
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
