#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS SYRK kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Syrk template is instantiated in the function CutlassSsyrkNN. This is kernel computes
  the symmetric rank-k update (SYRK) using double-precision doubleing-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 16x32x16 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the SSYRK kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for double-precision SYRK kernel
//

// Defines cutlass::gemm::device::Syrk, the generic Syrk computation template class.
#include "cutlass/gemm/device/rank_k.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS SYRK kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS SYRK template and launch a SYRK kernel.
hipError_t CutlassSsyrkNN(
  int N,
  int K,
  double alpha,
  double const *A,
  int lda,
  double beta,
  double *C,
  int ldc) {

  // Define type definition for double-precision CUTLASS SYRK with column-major
  // input matrices and 16x32x16 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for double-precision SYRK. Typical values are used as
  // default template arguments.
  //
  // To view the full syrk device API interface, see `cutlass/gemm/device/syrk.h`

  using ColumnMajor = cutlass::layout::ColumnMajor;

  using CutlassSyrk = cutlass::gemm::device::RankK<
    double,
    ColumnMajor,
    double,
    ColumnMajor,
    cutlass::FillMode::kLower,
    double,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<16, 32, 16>,
    cutlass::gemm::GemmShape<16, 16, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      double,
      1,
      double,
      double
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    5,     // Stages
    1,     // AligmentA
    false, // SplitKSerail
    cutlass::arch::OpMultiplyAdd, 
    cutlass::ComplexTransform::kNone, 
    cutlass::BlasMode::kSymmetric
  >;

  // Define a CUTLASS SYRK type
  CutlassSyrk syrk_operator;

  // Construct the CUTLASS SYRK arguments object.
  //
  // One of CUTLASS's design patterns is to define syrk argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Syrk and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassSyrk::Arguments args(cutlass::gemm::GemmUniversalMode::kGemm,
                              {N, N, K}, // Syrk Problem dimensions
                              1, // batch_count,
                              {alpha, beta}, // Scalars used in the Epilogue
                              reinterpret_cast<void const *>(A),
                              const_cast<void *>(reinterpret_cast<void *>(C)),
                              reinterpret_cast<void *>(C), // destination matrix D (may be different memory than source C matrix)
                              (int64_t)N*K, // Batch strides
                              (int64_t)N*N,
                              (int64_t)N*N,
                              lda,
                              ldc,
                              ldc);

  //
  // Launch the CUTLASS SYRK kernel.
  //
  
  cutlass::Status status = syrk_operator(args);

  //
  // Return a hipError_t if the CUTLASS SYRK operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  double *matrix,
  int ldm,
  int rows,
  int columns,
  int seed = 0) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    int offset = i + j * ldm;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    double value = double(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(double *matrix, int ldm, int rows, int columns, int seed = 0) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns, seed);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(double **matrix, int ldm, int rows, int columns, int seed = 0) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(double) * ldm * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  result = InitializeMatrix(*matrix, ldm, rows, columns, seed);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Naive reference SYRK computation.
__global__ void ReferenceSyrk_kernel(
  int N,
  int K,
  double alpha,
  double const *A,
  int lda,
  double beta,
  double *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < N && j < N && i >= j ) { // Since C is in Lower Fill Mode
    double accumulator = 0;

    for (int k = 0; k < K; ++k) {
      accumulator += A[i + k * lda] * A[j + k * lda];
    }

    C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
  }
}

/// Reference SYRK computation.
hipError_t ReferenceSyrk(
  int N,
  int K,
  double alpha,
  double const *A,
  int lda,
  double beta,
  double *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (N + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceSyrk_kernel<<< grid, block >>>(N, K, alpha, A, lda, beta, C, ldc);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a double-precision
/// CUTLASS SYRK kernel.
hipError_t TestCutlassSyrk(int N, int K, double alpha, double beta) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to SYRK kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = N;
  int ldc = N;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(double) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  double *A;
  double *C_cutlass;
  double *C_reference;

  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  result = AllocateMatrix(&A, lda, N, K, 0);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateMatrix(&C_cutlass, ldc, N, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_reference, ldc, N, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(C_cutlass);
    return result;
  }

  result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(A);

    return result;
  }

  //
  // Launch CUTLASS SYRK.
  //

  result = CutlassSsyrkNN(N, K, alpha, A, lda, beta, C_cutlass, ldc);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS SYRK kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(A);

    return result;
  }

  //
  // Verify.
  //

  // Launch reference SYRK
  result = ReferenceSyrk(N, K, alpha, A, lda, beta, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference SYRK kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<double> host_cutlass(ldc * N, 0);
  std::vector<double> host_reference(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS SYRK results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference SYRK results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(A);

    return result;
  }

  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C_cutlass);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  }

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_syrk example.
//
// usage:
//
//   00_basic_syrk <N> <K> <alpha> <beta>
//
int main(int argc, const char *arg[]) {

  bool notSupported = false;

  // CUTLASS must be compiled with CUDA 11 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "NVIDIA  Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;

    return -1;
  }

  if (!((props.major * 10 + props.minor) >= 80)) {

    std::cerr << "This example requires compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  //
  // Parse the command line to obtain SYRK dimensions and scalar values.
  //

  // SYRK problem dimensions.
  int problem[2] = { 128, 128 };

  for (int i = 1; i < argc && i < 3; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  double scalars[2] = { 1, 0 };

  for (int i = 3; i < argc && i < 5; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 3];
  }

  //
  // Run the CUTLASS SYRK test.
  //

  hipError_t result = TestCutlassSyrk(
    problem[0],     // SYRK N dimension
    problem[1],     // SYRK K dimension
    scalars[0],     // alpha
    scalars[1]      // beta
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
