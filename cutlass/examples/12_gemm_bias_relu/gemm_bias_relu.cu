#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
*/

#include <algorithm>
#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/epilogue/thread/linear_combination_relu.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"
#include "helper.h"

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = float;                   // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = cutlass::half_t;              // <- data type of elements in input matrix A
using ElementInputB = cutlass::half_t;              // <- data type of elements in input matrix B
using ElementOutput = float;                        // <- data type of elements in output matrix D

// The code section below describes matrix layout of input and output matrices.
// Column Major for Matrix A, B and C.

// Note that if the output is column major, the bias has to be per row. i.e. every row has different bias.
// If the output is row major, the bias has to be per column, i.e. every column has different bias.
// Below list some other notices:
//   1) we only have row major epilogue.
//   2) we swap A and B if the output is column major then we can still use the
//      row major epilogue.
//   3) Mx1 bias vector becomes 1xM after the swapping/transposing.
//   4) we can use the existing OutputIterator to load 1xM bias vector.

using LayoutInputA = cutlass::layout::ColumnMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::ColumnMajor;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm75;

// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock =
    cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32 
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 8>;  // <- MMA Op tile M = 8, N = 8, K = 4

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??

// Define the epilogue operation as LinearCombinationRelu. This is approximately equal to
//
//    d_ij = max(0, alpha * sum_k(a_ik * b_kj) + c_ij )
//
using EpilogueOp = cutlass::epilogue::thread::LinearCombinationRelu<
    ElementOutput,                                        // <- data type of output matrix
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // <- this is the number of elements per
                                                          // vectorized memory access. For half
                                                          // precision, it's 8 elements. This becomes
                                                          // the vector width of math instructions in
                                                          // epilogue too
    ElementAccumulator,                                   // <- data type of accumulator
    ElementComputeEpilogue,                               // <- data type for alpha in linear combination function
    cutlass::epilogue::thread::ScaleType::NoBetaScaling>; // <- alpha x C + bias

// Number of pipelines you want to use
constexpr int NumStages = 2;

using Gemm = cutlass::gemm::device::Gemm<ElementInputA,
                                         LayoutInputA,
                                         ElementInputB,
                                         LayoutInputB,
                                         ElementOutput,
                                         LayoutOutput,
                                         ElementAccumulator,
                                         MMAOp,
                                         SmArch,
                                         ShapeMMAThreadBlock,
                                         ShapeMMAWarp,
                                         ShapeMMAOp,
                                         EpilogueOp,
                                         SwizzleThreadBlock,
                                         NumStages>;

int run() {

  const int length_m = 5120;
  const int length_n = 4096;
  const int length_k = 4096;

  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size(length_m, length_n, length_k);

  // Initialize tensors using CUTLASS helper functions
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(
      problem_size.mk());  // <- Create matrix A with dimensions M x K
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(
      problem_size.kn());  // <- Create matrix B with dimensions K x N

  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c_bias(
      {problem_size.m(), 1});  // <- Create matrix C with dimensions M x 1

  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // CUTLASS kernel
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // reference kernel

  // Fill input and output matrices on host using CUTLASS helper functions
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(4),
      ElementInputA(-4),
      0);  // <- Fill matrix A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(4),
      ElementInputB(-4),
      0);  // <- Fill matrix B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c_bias.host_view(),
      1,
      ElementOutput(4),
      ElementOutput(-4),
      0);  // <- Fill matrix C on host with uniform-distribution random data
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- fill matrix D on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- fill matrix D for reference on host with zeros

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c_bias.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  // Initialize alpha for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename Gemm::Arguments arguments{
    problem_size,                       // <- problem size of matrix multiplication
    tensor_a.device_ref(),              // <- reference to matrix A on device
    tensor_b.device_ref(),              // <- reference to matrix B on device

    {tensor_c_bias.device_data(), 0},   // <- the C matrix is treated as the bias vector. We can enable the GEMM
                                        //    to project away the N dimension by setting the stride to zero.

    tensor_d.device_ref(),              // <- reference to matrix D on device
    {alpha},                              // <- alpha
    split_k_slices};                    // <- k-dimension split factor

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm_op;

  // Check the problem size is supported or not 
  cutlass::Status status = gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(status);

  // Launch initialized CUTLASS kernel
  status = gemm_op();
  CUTLASS_CHECK(status);

  //
  // Create instantiation for device reference gemm kernel
  //

  cutlass::reference::device::Gemm<ElementInputA,
                                   LayoutInputA,
                                   ElementInputB,
                                   LayoutInputB,
                                   ElementOutput,
                                   LayoutOutput,
                                   ElementComputeEpilogue,
                                   ElementComputeEpilogue>
      gemm_device_reference;

  // Launch device reference to compute strictly the product A * B
  gemm_device_reference(
    problem_size,
    alpha,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    0,
    tensor_ref_d.device_ref());

  // Wait for kernels to finish
  hipDeviceSynchronize();

  // Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d.sync_host();
  tensor_ref_d.sync_host();

  // Compute bias + relu in host code
  for (int i = 0; i < problem_size.m(); ++i) {
    for (int j = 0; j < problem_size.n(); ++j) {
      tensor_ref_d.at({i, j}) = std::max(
        ElementOutput(0), 
        ElementOutput(tensor_ref_d.at({i, j}) + tensor_c_bias.at({i, 0}))
      );
    }
  }

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  std::cout << (cutlass::reference::host::TensorEquals(tensor_d.host_view(),
                                                       tensor_ref_d.host_view())
                    ? "Passed"
                    : "Failed")
            << std::endl;

  CUTLASS_CHECK(status);
  return 0;
}

int main() {

  bool notSupported = false;

  // Turing Tensor Core operations exposed with mma.sync are first available in CUDA 10.2.
  //
  // CUTLASS must be compiled with CUDA 10.1 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
    std::cerr << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!(props.major * 10 + props.minor >= 75)) {
    std::cerr << "Turing Tensor Ops must be run on a machine with compute capability at least 75."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  return run();
}
