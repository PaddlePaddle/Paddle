#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example demonstrates how to call a CUTLASS TRMM kernel and provides a naive reference
  matrix multiply kernel to verify its correctness.

  The CUTLASS Trmm template is instantiated in the function CutlassStrmmNN. This is kernel computes
  the triangular matrix product (TRMM) using double-precision doubleing-point arithmetic and assumes
  all matrices have column-major layout.

  The threadblock tile size is chosen as 64x64x16 which offers good performance for large matrices.
  See the CUTLASS Parallel for All blog post for more exposition on the tunable parameters available
  in CUTLASS.

  https://devblogs.nvidia.com/cutlass-linear-algebra-cuda/

  Aside from defining and launching the STRMM kernel, this example does not use any other components
  or utilities within CUTLASS. Such utilities are demonstrated elsewhere in other examples and are
  prevalent in the CUTLASS unit tests.

*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

// Helper methods to check for errors
#include "helper.h"

//
// CUTLASS includes needed for double-precision TRMM kernel
//

// Defines cutlass::gemm::device::Trmm, the generic Trmm computation template class.
#include "cutlass/gemm/device/trmm.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS TRMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Define a CUTLASS TRMM template and launch a TRMM kernel.
hipError_t CutlassStrmmNN(
  int M,
  int N,
  double alpha,
  double const *A,
  int lda,
  double const *B,
  int ldb,
  double *C,
  int ldc) {

  // Define type definition for double-precision CUTLASS TRMM with column-major
  // input matrices and 64x64x16 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for double-precision TRMM. Typical values are used as
  // default template arguments.
  //
  // To view the full trmm device API interface, see `cutlass/gemm/device/trmm.h`

  using ColumnMajor = cutlass::layout::ColumnMajor;

  using CutlassTrmm = cutlass::gemm::device::Trmm<
    double,
    ColumnMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kLower,
    cutlass::DiagType::kNonUnit,
    double,
    ColumnMajor,
    double,
    ColumnMajor,
    double,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      double,
      1,
      double,
      double,
      cutlass::epilogue::thread::ScaleType::OnlyAlphaScaling 
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    5,
    1,
    1,
    false,
    cutlass::arch::OpMultiplyAdd
  >;

  // Define a CUTLASS TRMM type
  CutlassTrmm trmm_operator;

  // Construct the CUTLASS TRMM arguments object.
  //
  // One of CUTLASS's design patterns is to define trmm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Trmm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassTrmm::Arguments args(cutlass::gemm::GemmUniversalMode::kGemm,
                              {M, N, M}, // Trmm Problem dimensions in Left-Side Mode
                              1, // batch_count,
                              {alpha}, // Scalars used in the Epilogue
                              reinterpret_cast<void const *>(A),
                              reinterpret_cast<void const *>(B),
                              reinterpret_cast<void *>(C), // destination matrix D (may be different memory than source C matrix)
                              (int64_t)M*M, // Batch strides
                              (int64_t)M*N,
                              (int64_t)M*N,
                              lda,
                              ldb,
                              ldc);

  //
  // Launch the CUTLASS TRMM kernel.
  //
  
  cutlass::Status status = trmm_operator(args);

  //
  // Return a hipError_t if the CUTLASS TRMM operator returned an error code.
  //

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel to initialize a matrix with small integers.
__global__ void InitializeMatrix_kernel(
  double *matrix,
  int ldm,
  int rows,
  int columns,
  int seed = 0,
  cutlass::FillMode fill_mode = cutlass::FillMode::kInvalid) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < rows && j < columns) {
    if (fill_mode == cutlass::FillMode::kLower && i < j) return;
    else if (fill_mode == cutlass::FillMode::kUpper && i > j) return;
    int offset = i + j * ldm;

    // Generate arbitrary elements.
    int const k = 16807;
    int const m = 16;
    double value = double(((offset + seed) * k % m) - m / 2);

    matrix[offset] = value;
  }
}

/// Simple function to initialize a matrix to arbitrary small integers.
hipError_t InitializeMatrix(double *matrix, int ldm, int rows, int columns, int seed = 0,
                             cutlass::FillMode fill_mode = cutlass::FillMode::kInvalid) {

  dim3 block(16, 16);
  dim3 grid(
    (rows + block.x - 1) / block.x,
    (columns + block.y - 1) / block.y
  );

  InitializeMatrix_kernel<<< grid, block >>>(matrix, ldm, rows, columns, seed, fill_mode);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates device memory for a matrix then fills with arbitrary small integers.
hipError_t AllocateMatrix(double **matrix, int ldm, int rows, int columns, int seed = 0,
                           cutlass::FillMode fill_mode = cutlass::FillMode::kInvalid) {
  hipError_t result;

  size_t sizeof_matrix = sizeof(double) * ldm * columns;

  // Allocate device memory.
  result = hipMalloc(reinterpret_cast<void **>(matrix), sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to allocate matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Clear the allocation.
  result = hipMemset(*matrix, 0, sizeof_matrix);

  if (result != hipSuccess) {
    std::cerr << "Failed to clear matrix device memory: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  // Initialize matrix elements to arbitrary small integers.
  result = InitializeMatrix(*matrix, ldm, rows, columns, seed, fill_mode);

  if (result != hipSuccess) {
    std::cerr << "Failed to initialize matrix: "
      << hipGetErrorString(result) << std::endl;
    return result;
  }

  return result;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Naive reference TRMM computation.
__global__ void ReferenceTrmm_kernel(
  int M,
  int N,
  double alpha,
  double const *A,
  int lda,
  double const *B,
  int ldb,
  double *C,
  int ldc) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < M && j < N) {
    double accumulator = 0;

    for (int k = 0; k < M; ++k) {
      accumulator += A[i + k * lda] * B[k + j * ldb]; // Since A is in Left-Side Mode
    }

    C[i + j * ldc] = alpha * accumulator;
  }
}

/// Reference TRMM computation.
hipError_t ReferenceTrmm(
  int M,
  int N,
  double alpha,
  double const *A,
  int lda,
  double const *B,
  int ldb,
  double *C,
  int ldc) {

  dim3 block(16, 16);
  dim3 grid(
    (M + block.x - 1) / block.x,
    (N + block.y - 1) / block.y
  );

  ReferenceTrmm_kernel<<< grid, block >>>(M, N, alpha, A, lda, B, ldb, C, ldc);

  return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocate several matrices in GPU device memory and call a double-precision
/// CUTLASS TRMM kernel.
hipError_t TestCutlassTrmm(int M, int N, double alpha) {
  hipError_t result;

  //
  // Define several matrices to be used as operands to TRMM kernels.
  //

  // Compute leading dimensions for each matrix.
  int lda = M;
  int ldb = M;
  int ldc = M;

  // Compute size in bytes of the C matrix.
  size_t sizeof_C = sizeof(double) * ldc * N;

  // Define pointers to matrices in GPU device memory.
  double *A;
  double *B;
  double *C_cutlass;
  double *C_reference;

  //
  // Allocate matrices in GPU device memory with arbitrary seeds.
  //

  result = AllocateMatrix(&A, lda, M, M, 0, cutlass::FillMode::kLower);

  if (result !=  hipSuccess) {
    return result;
  }

  result = AllocateMatrix(&B, ldb, M, N, 17);

  if (result !=  hipSuccess) {
    hipFree(A);
    return result;
  }

  result = AllocateMatrix(&C_cutlass, ldc, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    return result;
  }

  result = AllocateMatrix(&C_reference, ldc, M, N, 101);

  if (result != hipSuccess) {
    hipFree(A);
    hipFree(B);
    hipFree(C_cutlass);
    return result;
  }

  result = hipMemcpy(C_reference, C_cutlass, sizeof_C, hipMemcpyDeviceToDevice);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy C_cutlass matrix to C_reference: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Launch CUTLASS TRMM.
  //

  result = CutlassStrmmNN(M, N, alpha, A, lda, B, ldb, C_cutlass, ldc);

  if (result != hipSuccess) {
    std::cerr << "CUTLASS TRMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Verify.
  //

  // Launch reference TRMM
  result = ReferenceTrmm(M, N, alpha, A, lda, B, ldb, C_reference, ldc);

  if (result != hipSuccess) {
    std::cerr << "Reference TRMM kernel failed: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  // Copy to host and verify equivalence.
  std::vector<double> host_cutlass(ldc * N, 0);
  std::vector<double> host_reference(ldc * N, 0);

  result = hipMemcpy(host_cutlass.data(), C_cutlass, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy CUTLASS TRMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  result = hipMemcpy(host_reference.data(), C_reference, sizeof_C, hipMemcpyDeviceToHost);

  if (result != hipSuccess) {
    std::cerr << "Failed to copy Reference TRMM results: "
      << hipGetErrorString(result) << std::endl;

    hipFree(C_reference);
    hipFree(C_cutlass);
    hipFree(B);
    hipFree(A);

    return result;
  }

  //
  // Free device memory allocations.
  //

  hipFree(C_reference);
  hipFree(C_cutlass);
  hipFree(B);
  hipFree(A);

  //
  // Test for bit equivalence of results.
  //

  if (host_cutlass != host_reference) {
    std::cerr << "CUTLASS results incorrect." << std::endl;

    return hipErrorUnknown;
  }

  return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to basic_trmm example.
//
// usage:
//
//   00_basic_trmm <M> <N> <alpha> 
//
int main(int argc, const char *arg[]) {

  bool notSupported = false;

  // CUTLASS must be compiled with CUDA 11 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "NVIDIA  Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;

    return -1;
  }

  if (!((props.major * 10 + props.minor) >= 80)) {

    std::cerr << "This example requires compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  //
  // Parse the command line to obtain TRMM dimensions and scalar values.
  //

  // TRMM problem dimensions.
  int problem[2] = { 128, 128 };

  for (int i = 1; i < argc && i < 3; ++i) {
    std::stringstream ss(arg[i]);
    ss >> problem[i - 1];
  }

  // Scalars used for linear scaling the result of the matrix product.
  double scalars[1] = { 1 };

  for (int i = 3; i < argc && i < 4; ++i) {
    std::stringstream ss(arg[i]);
    ss >> scalars[i - 3];
  }

  //
  // Run the CUTLASS TRMM test.
  //

  hipError_t result = TestCutlassTrmm(
    problem[0],     // TRMM M dimension
    problem[1],     // TRMM N dimension
    scalars[0]     // alpha
  );

  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
