/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
This example shows how to compute conv2d gradient with respect to weight (wgrad). In wgrad, the K dimension of
impligit GEMM, corresponding to the sequential reduction loop, is very large (N * P * Q). Split-k with parallel
reduction is highly effective for such cases. Given split_k_slices parameter, it partitions the K loop into
split_k_slices chunks and computes partial reductions in parallel across different blocks. After that,
a parallel reduction kernel is launched to accumulate partial reductions.
In practice, wgrad requires fp32 accumulation to avoid overflow. When the input is fp16, some care is needed
to correctly instantiate the GEMM template.
*/

#include <iostream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_wgrad.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/convolution.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/reduction/device/reduce_split_k.h"
#include "cutlass/reduction/thread/reduction_operators.h"

#include "helper.h"

// The code section below describes datatype for input, output tensors and computation between
// elements
// In Wgrad, fp32 accumulation is necessary in practice.
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;             // Data type of elements in input tensor
using ElementOutput = cutlass::half_t;                       // Data type of elements in output tensor
using ElementC = ElementOutput;
using ElementCompute = ElementComputeEpilogue;
using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>; // Threadblock tile shape

// This code section describes tile size a warp will compute
using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;          // Warp tile shape

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;    // TensorCore instruction shape

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipelines you want to use
constexpr int NumStages = 3;

// This code section describe iterator algorithm selected is Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// We need two epilogue functors - one for GEMM and another for the final reduction.
// The epilogue for GEMM is not used, but needed to instantiate the CUTLASS kernel template.
// Note that, when the input is fp16 and accumulation is fp32, the output of GEMM needs to be fp32,
// the final reduction is done in fp32, and the reduction epilogue converts fp32 outputs to fp16.
// Therefore, the output type of the GEMM epilogue is ElementCompute, not ElementOutput.

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOpGEMM = cutlass::epilogue::thread::LinearCombination<
    ElementCompute,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementCompute>::value,  // The number of elements per vectorized.
    // memory access. This becomes the vector width of
    // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

// The epilogue functor for reduction. This is the one that is actually used.
using EpilogueOpReduction = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // The number of elements per vectorized.
    // memory access. This becomes the vector width of
    // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in lin

using Conv2dWgradKernel = typename cutlass::conv::kernel::DefaultConv2dWgrad<
    ElementInputA, LayoutInputA,
    ElementInputB, LayoutInputB,
    ElementAccumulator, LayoutOutput,
    ElementAccumulator,
    MMAOp,
    SmArch,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOpGEMM,
    SwizzleThreadBlock,
    NumStages,
    cutlass::arch::OpMultiplyAdd,
    IteratorAlgorithm
    >::Kernel;

using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dWgradKernel>;

using EpilogueOutputOp = EpilogueOpReduction;

/// Reduction kernel
using ReductionOp = cutlass::reduction::thread::ReduceAdd<
    ElementAccumulator,
    typename EpilogueOutputOp::ElementAccumulator,
    EpilogueOutputOp::kCount
   >;

using ReductionKernel = cutlass::reduction::kernel::ReduceSplitK<
    cutlass::MatrixShape<4, 32 * EpilogueOutputOp::kCount>,
    EpilogueOutputOp,
    ReductionOp
   >;

using ReductionDevice = cutlass::reduction::device::ReduceSplitK<ReductionKernel>;
using ReductionStrideIndex = typename ReductionDevice::StrideIndex;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  int split_k_slices;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    reference_check(true),
    measure_performance(false),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    split_k_slices(8),
    benchmark(false) { }

  // Verify the problem size is compatible with the CUTLASS Convolution implementation.
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((input_size.c() % kAlignment) ||
	(filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
	(padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Updates input and filter sizes
  void update(
	      cutlass::Tensor4DCoord input_size,
	      cutlass::Tensor4DCoord filter_size,
	      cutlass::MatrixCoord stride) {

    this->input_size = input_size;
    this->filter_size = filter_size;
    conv_stride = stride;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c();

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("split-k-slices", split_k_slices);

    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "30_wgrad_split_k example\n\n"
	<< "  This example shows how to compute conv2d gradient with respect to weight (wgrad).\n"
	<< "  In wgrad, the K dimension of impligit GEMM, corresponding to the sequential reduction loop, is very large (N * P * Q).\n"
	<< "  Split-k with parallel reduction is highly effective for such cases.\n\n"
	<< "Options:\n\n"
	<< "  --help               If specified, displays this usage statement.\n\n"
	<< "  --n=<int>            Input tensor extent N\n"
	<< "  --h=<int>            Input tensor extent H\n"
	<< "  --w=<int>            Input tensor extent W\n"
	<< "  --c=<int>            Input tensor extent C\n"
	<< "  --k=<int>            Filter extent K\n"
	<< "  --r=<int>            Filter extent R\n"
	<< "  --s=<int>            Filter extent S\n\n"
	<< "  --alpha=<float>      Epilogue scalar alpha\n"
	<< "  --beta=<float>       Epilogue scalar beta\n\n"
	<< "  --split-k-slices=<int>   Split-k factor \n\n"
	<< "  --ref-check          If set (true), reference check on the host is computed\n"
	<< "  --perf-check         If set (true), performance is measured.\n"
	<< "  --benchmark          If set (true), performance benchmarking on several layers and batch-size.\n"
	<< "  --iterations=<int>   Number of profiling iterations to perform.\n"
	<< "  --save-workspace     If set, workspace is written to a text file.\n"
	<< "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
	<< "$ ./examples/30_wgrad_split_k/30_wgrad_split_k --n=32 --h=224 --w=224 --c=128 --k=256 --r=3 --s=3 --split-k-slices=8\n\n";

    return out;
  }

  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(input_size.n(),
				  (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
				  (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
				  filter_size.n());
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());

    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result():
    runtime_ms(0),
    gflops(0),
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,Stride_H,Stride_W,Runtime,GFLOPs";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << options.conv_stride.row() << ","
      << options.conv_stride.column() << ","
      << runtime_ms << ","
      << gflops;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Runs one benchmark
Result profile_convolution(Options const &options) {

  Result result;

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  // Inputs are the output gradient and the original activation.
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.output_size());
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.input_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.filter_size);

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8),
      0);

  // Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8),
      0);

  // Fill tensor C, D on host with zeros
  cutlass::reference::host::TensorFill(tensor_c.host_view());

  cutlass::reference::host::TensorFill(tensor_d.host_view());

  // Fill tensor D for reference on host with zeros
  cutlass::reference::host::TensorFill(tensor_ref_d.host_view());

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Partition the GEMM K loop into split_k_slices chunks
  int split_k_slices = options.split_k_slices;

  // Construct Conv2dProblemSize with user defined output size
  // Do not forget to pass the last argument.
  cutlass::conv::Conv2dProblemSize problem_size(
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices
  );

  using cutlass::layout::TensorNHWC;

  cutlass::conv::SplitKMode const split_k_mode = cutlass::conv::SplitKMode::kParallel;

  // Since the epilogue is not computed after GEMM, there is no need to pass the C tensor and
  // alpha and beta can be set to 1 and 0 respectively.
  // Moreover, since the output will be written to the workspace, there is no need to pass
  // the D tensor as well.
  // Do not forget to pass the last argument.
  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    {nullptr, TensorNHWC()},
    {nullptr, TensorNHWC()},
    {ElementCompute(1), ElementCompute(0)},
    split_k_mode
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemm implicit_gemm;

  size_t workspace_size = implicit_gemm.get_workspace_size(arguments);

  // Split-K requires non-zero workspace size. The workspace size grows linearly with split_k_slices.
  std::cout << "split-k-slices: " << split_k_slices << std::endl;
  std::cout << "workspace size: " << workspace_size << std::endl;

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  // After the workspace is allocated, we point the GEMM destination pointer to the workspace.
  TensorNHWC layout_D{TensorNHWC::packed(options.filter_size)};
  arguments.ref_D.reset(reinterpret_cast<ElementCompute*>(workspace.get()), layout_D);

  result.status = implicit_gemm.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm();

  CUTLASS_CHECK(result.status);

  if (split_k_mode == cutlass::conv::SplitKMode::kParallel) {
    // Do reduction
    ReductionDevice reduction_op;
    auto& status = result.status;
    static cutlass::conv::Operator const kConvolutionalOperator = ImplicitGemm::kConvolutionalOperator;
    typename ReductionDevice::Arguments reduction_args(
        cutlass::conv::implicit_gemm_problem_size(kConvolutionalOperator, problem_size).mn(),
        problem_size.split_k_slices,
        cutlass::conv::implicit_gemm_tensor_c_size(kConvolutionalOperator, problem_size),
        // Reduction input
        {
            reinterpret_cast<ElementAccumulator*> (workspace.get()),
            ReductionStrideIndex(tensor_c.stride()[ImplicitGemm::ImplicitGemmKernel::kTensorCStrideIdx])
        },
        // Destination
        {
            tensor_d.device_data(),
            ReductionStrideIndex(tensor_d.stride()[ImplicitGemm::ImplicitGemmKernel::kTensorCStrideIdx])
        },
        // Source
        {
            tensor_c.device_data(),
            ReductionStrideIndex(tensor_c.stride()[ImplicitGemm::ImplicitGemmKernel::kTensorCStrideIdx])
        },
        {options.alpha, options.beta}
    );

    status = reduction_op.initialize(reduction_args, nullptr);
    status = reduction_op();
  }

  //
  // Optional reference check
  //

  if (options.reference_check) {
    std::cout << "Verification on device...\n";

    // Compute with reference implementation
    cutlass::reference::device::Conv2dWgrad<
      ElementInputA,
	LayoutInputA,
	ElementInputB,
	LayoutInputB,
	ElementOutput,
	LayoutOutput,
	ElementComputeEpilogue,
	ElementAccumulator,
	cutlass::NumericConverter<ElementOutput, ElementComputeEpilogue>
	>(
	  problem_size,
	  tensor_a.device_ref(),
	  tensor_b.device_ref(),
	  tensor_c.device_ref(),
	  tensor_ref_d.device_ref(),
	  options.alpha,
	  options.beta
	  );

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    tensor_c.sync_host();
    tensor_d.sync_host();
    tensor_ref_d.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(tensor_d.host_view(), tensor_ref_d.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    }
    else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  }
  else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "26_ampere_fused_wgrad_batch_normalization_"
       << options.input_size.n() << "x" << options.input_size.h() << "x" << options.input_size.w() << "x" << options.input_size.c()
       << "_"
       << options.filter_size.n() << "x" << options.filter_size.h() << "x" << options.filter_size.w() << "x" << options.filter_size.c()
       << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace
      << "Input = \n" << tensor_a.host_view() << "\n\n"
      << "Filters = \n" << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference = \n" << tensor_ref_d.host_view() << "\n\n";
    }

    output_workspace << "Computed = \n" << tensor_c.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }

  //
  // Performance measurement
  //

  if (options.measure_performance) {

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // Record an event at the start of a series of convolution operations.
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Launch a sequence of implicit GEMM operations on the device
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm();
      CUTLASS_CHECK(result.status);
    }

    // Record an event when the convolutions have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Print average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major > 8 || (props.major == 8 && props.minor >= 0))) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // Benchmark several layers

    int batch_sizes[] = {34, 408};

    struct Benchmark {
      int h, w, c, k, r, s, stride_h, stride_w;
    } layers[] = {
	 {56, 56,   64,  256, 1, 1, 1, 1},
	 {56, 56,   64,   64, 1, 1, 1, 1},
	 {56, 56,   64,   64, 3, 3, 1, 1},
	 {56, 56,  256,   64, 1, 1, 1, 1},
	 {56, 56,  256,  512, 1, 1, 2, 2},
	 {56, 56,  256,  128, 1, 1, 1, 1},
	 {56, 56,  128,  128, 3, 3, 2, 2},
	 {28, 28,  128,  512, 1, 1, 1, 1},
	 {28, 28,  512,  128, 1, 1, 1, 1},
	 {28, 28,  128,  128, 3, 3, 1, 1},
	 {28, 28,  512, 1024, 1, 1, 2, 2},
	 {28, 28,  512,  256, 1, 1, 1, 1},
	 {28, 28,  256,  256, 3, 3, 2, 2},
	 {14, 14,  256, 1024, 1, 1, 1, 1},
	 {14, 14, 1024,  256, 1, 1, 1, 1},
	 {14, 14,  256,  256, 3, 3, 1, 1},
	 {14, 14, 1024, 2048, 1, 1, 2, 2},
	 {14, 14, 1024,  512, 1, 1, 1, 1},
	 {14, 14,  512,  512, 3, 3, 2, 2},
	 { 7,  7,  512, 2048, 1, 1, 1, 1},
	 { 7,  7, 2048,  512, 1, 1, 1, 1},
	 { 7,  7,  512,  512, 3, 3, 1, 1},
    };

    Result::print_header(std::cout, options) << std::endl;

    int idx = 1;

    for (auto const &layer : layers) {
      for (auto N : batch_sizes) {
        options.update({N, layer.h, layer.w, layer.c},
                       {layer.k, layer.r, layer.s, layer.c},
                       {layer.stride_h, layer.stride_w});

        Result result = profile_convolution(options);
        result.print(std::cout, idx, options) << std::endl;
      }

      ++idx;
    }
  }
  else {

    // Execute one problem size
    if (!options.valid()) {
      std::cerr << "Invalid problem." << std::endl;
      return -1;
    }

    Result result = profile_convolution(options);

    Result::print_header(std::cout, options) << std::endl;
    result.print(std::cout, 1, options) << std::endl;
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
