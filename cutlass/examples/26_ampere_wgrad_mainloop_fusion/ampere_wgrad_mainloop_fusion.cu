#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**

This example shows how to fuse activation's per channel scale+bias+relu 
into the wgrad mainloop.

Compared with original fprop kernel, this example has two more vectors, one for
the scale and one for the bias.  The length of the vectors are the same as the
activation channel number.  This kernels loads the vectors when the associated
activation channels are loaded in the mainloop.  Between reading the 
activations and scale/bias data from the shared memory and calling tensor core
instructions, scale+bias+relu is computed in the register file.

This example is customized for Ampere 16816 fp16 tensor core instruction.
Changing to different data types or different tensor core instruction require
source code changing.  See
include/cutlass/conv/threadblock/implicit_gemm_wgrad_fusion_multistage.h for more
technical details.
*/

#include <iostream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_wgrad_fusion.h"
#include "cutlass/conv/device/implicit_gemm_convolution_fusion.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

// The code section below describes datatype for input, output tensors and computation between
// elements 
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputScaleBias = cutlass::half_t;     // Data type of elements in input sclae and bias vectors
using ElementOutput = float;                       // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutInputScaleBias = cutlass::layout::RowMajor;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>; // Threadblock tile shape

// This code section describes tile size a warp will compute
using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;          // Warp tile shape

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;    // TensorCore instruction shape

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipelines you want to use
constexpr int NumStages = 5;

// This code section describe iterator algorithm selected is Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // The number of elements per vectorized.
                                                       // memory access. This becomes the vector width of
                                                       // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

using Conv2dWgradFusionKernel = typename cutlass::conv::kernel::DefaultConv2dWgradFusion<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementInputScaleBias, LayoutInputScaleBias,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  IteratorAlgorithm
>::Kernel;

using ImplicitGemmFusion = cutlass::conv::device::ImplicitGemmConvolutionFusion<Conv2dWgradFusionKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    reference_check(true),
    measure_performance(false),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    benchmark(false) { }

  // Verify the problem size is compatible with the CUTLASS Convolution implementation.
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Updates input and filter sizes
  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size,
    cutlass::MatrixCoord stride) {

    this->input_size = input_size;
    this->filter_size = filter_size;
    conv_stride = stride;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c(); 

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "26_ampere_fused_wgrad_batch_normalization example\n\n"
      << "  This example fuses scale+bias+relu from batch norm into Ampere's\n"
      << "  Tensor Core operators on F16 data types to compute\n"
      << "  backward convolution on tensors of layout NHWC.\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --n=<int>            Input tensor extent N\n"
      << "  --h=<int>            Input tensor extent H\n"
      << "  --w=<int>            Input tensor extent W\n"
      << "  --c=<int>            Input tensor extent C\n"
      << "  --k=<int>            Filter extent K\n"
      << "  --r=<int>            Filter extent R\n"
      << "  --s=<int>            Filter extent S\n\n"
      << "  --alpha=<float>      Epilogue scalar alpha\n"
      << "  --beta=<float>       Epilogue scalar beta\n\n"
      << "  --ref-check          If set (true), reference check on the host is computed\n"
      << "  --perf-check         If set (true), performance is measured.\n"
      << "  --benchmark          If set (true), performance benchmarking on several layers and batch-size.\n"
      << "  --iterations=<int>   Number of profiling iterations to perform.\n"
      << "  --save-workspace     If set, workspace is written to a text file.\n"
      << "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/26_ampere_fused_fprop_batch_normalization/26_ampere_fused_wgrad_batch_normalization  --n=32 --h=224 --w=224 --c=128 --k=256 --r=1 --s=1\n\n"
      << "$ ./examples/26_ampere_fused_fprop_batch_normalization/26_ampere_fused_wgrad_batch_normalization  --n=1 --h=224 --w=224 --c=32 --k=32 --r=3 --s=3 --ref-check\n\n";

    return out;
  }
  
  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result(): 
    runtime_ms(0), 
    gflops(0),
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,Stride_H,Stride_W,Runtime,GFLOPs";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out 
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << options.conv_stride.row() << ","
      << options.conv_stride.column() << ","
      << runtime_ms << ","
      << gflops;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Runs one benchmark
Result profile_convolution(Options const &options) {

  Result result;

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.output_size());
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.input_size);
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_transformed_b(options.input_size);
  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>
      tensor_b_scale({1, options.input_size.c()});
  cutlass::HostTensor<ElementInputScaleBias, LayoutInputScaleBias>
      tensor_b_bias({1, options.input_size.c()});

  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.filter_size);

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(3),
      ElementInputA(-4),
      0);

  // Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8),
      0);

  // Fill scale vector for tensor B on host with uniform-distribution random
  // data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b_scale.host_view(),
      1,
      ElementInputA(3),
      ElementInputA(-4),
      0);

  // Fill bias vector for tensor B on host with uniform-distribution random
  // data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b_bias.host_view(),
      1,
      ElementInputA(3),
      ElementInputA(-4),
      0);

  // Fill tensor C on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_c.host_view());

  // Fill tensor D on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());

  // Fill tensor D for reference on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_b_scale.sync_device();
  tensor_b_bias.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(      
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices
  );

  typename ImplicitGemmFusion::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    tensor_b_scale.device_ref(),
    tensor_b_bias.device_ref(),
    tensor_c.device_ref(),
    tensor_d.device_ref(),
    {options.alpha, options.beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemmFusion implicit_gemm_fusion_op;

  size_t workspace_size = implicit_gemm_fusion_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm_fusion_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_fusion_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_fusion_op();

  CUTLASS_CHECK(result.status);

  //
  // Optional reference check
  //
  
  if (options.reference_check) {
    std::cout << "Verification on device...\n";

    // Compute scale + bias + relu in host code
    for (int n = 0; n < options.input_size.n(); ++n) {
      for (int h = 0; h < options.input_size.h(); ++h) {
        for (int w = 0; w < options.input_size.w(); ++w) {
          for (int c = 0; c < options.input_size.c(); ++c) {
            tensor_transformed_b.at({n, h, w, c}) = std::max(
                ElementOutput(0), ElementOutput(tensor_b.at({n, h, w, c}) *
                                                    tensor_b_scale.at({0, c}) +
                                                tensor_b_bias.at({0, c})));
          }
        }
      }
    }

    tensor_transformed_b.sync_device();

    // Compute with reference implementation
    cutlass::reference::device::Conv2dWgrad<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementComputeEpilogue,
      ElementAccumulator,
      cutlass::NumericConverter<ElementOutput, ElementComputeEpilogue>
    >(
      problem_size,
      tensor_a.device_ref(),
      tensor_transformed_b.device_ref(),
      tensor_c.device_ref(),
      tensor_ref_d.device_ref(),
      options.alpha,
      options.beta
    );

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    tensor_d.sync_host();
    tensor_ref_d.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(
      tensor_d.host_view(),
      tensor_ref_d.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    }
    else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  }
  else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "26_ampere_wgrad_mainloop_fusion_"
      << options.input_size.n() << "x" << options.input_size.h() << "x" << options.input_size.w() << "x" << options.input_size.c() 
      << "_"
      << options.filter_size.n() << "x" << options.filter_size.h() << "x" << options.filter_size.w() << "x" << options.filter_size.c() 
      << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace 
      << "Input = \n" << tensor_a.host_view() << "\n\n"
      << "Filters = \n" << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference = \n" << tensor_ref_d.host_view() << "\n\n";
    }

    output_workspace << "Computed = \n" << tensor_d.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }
 
  //
  // Performance measurement
  //

  if (options.measure_performance) {

    hipEvent_t events[2];
    
    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // Record an event at the start of a series of convolution operations.
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Launch a sequence of implicit GEMM operations on the device
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm_fusion_op();
      CUTLASS_CHECK(result.status);
    }

    // Record an event when the convolutions have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Print average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major == 8 && props.minor == 0)) {
    std::cerr << "This test must run on SM80 A100.\n";
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // Benchmark several layers

    int batch_sizes[] = {34, 408};

    struct Benchmark {
      int h, w, c, k, r, s, stride_h, stride_w;
    } layers[] = {
      {56, 56,   64,  256, 1, 1, 1, 1},
      {56, 56,   64,   64, 1, 1, 1, 1},
      {56, 56,   64,   64, 3, 3, 1, 1},
      {56, 56,  256,   64, 1, 1, 1, 1},
      {56, 56,  256,  512, 1, 1, 2, 2},
      {56, 56,  256,  128, 1, 1, 1, 1},
      {56, 56,  128,  128, 3, 3, 2, 2},
      {28, 28,  128,  512, 1, 1, 1, 1},
      {28, 28,  512,  128, 1, 1, 1, 1},
      {28, 28,  128,  128, 3, 3, 1, 1},
      {28, 28,  512, 1024, 1, 1, 2, 2},
      {28, 28,  512,  256, 1, 1, 1, 1},
      {28, 28,  256,  256, 3, 3, 2, 2},
      {14, 14,  256, 1024, 1, 1, 1, 1},
      {14, 14, 1024,  256, 1, 1, 1, 1},
      {14, 14,  256,  256, 3, 3, 1, 1},
      {14, 14, 1024, 2048, 1, 1, 2, 2},
      {14, 14, 1024,  512, 1, 1, 1, 1},
      {14, 14,  512,  512, 3, 3, 2, 2},
      { 7,  7,  512, 2048, 1, 1, 1, 1},
      { 7,  7, 2048,  512, 1, 1, 1, 1},
      { 7,  7,  512,  512, 3, 3, 1, 1},
    };

    Result::print_header(std::cout, options) << std::endl;

    int idx = 1;

    for (auto const &layer : layers) {
      for (auto N : batch_sizes) {
        options.update({N, layer.h, layer.w, layer.c},
                       {layer.k, layer.r, layer.s, layer.c},
                       {layer.stride_h, layer.stride_w});

        Result result = profile_convolution(options);
        result.print(std::cout, idx, options) << std::endl;
      }

      ++idx;
    }
  }
  else {

    // Execute one problem size
    if (!options.valid()) {
      std::cerr << "Invalid problem." << std::endl;
      return -1;
    }

    Result result = profile_convolution(options);

    Result::print_header(std::cout, options) << std::endl;
    result.print(std::cout, 1, options) << std::endl;
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
