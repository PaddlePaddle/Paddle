#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*
  This example requires NVIDIA Ampere GPU or later.
*/

// Standard Library includes
#include <iostream>
#include <sstream>
#include <vector>

// CUTLASS Includes
#include "cutlass/cutlass.h"
#include "cutlass/functional.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/warp/default_mma_tensor_op.h"
#include "cutlass/epilogue/warp/fragment_iterator_tensor_op.h"
#include "cutlass/epilogue/warp/tile_iterator_tensor_op.h"

// CUTLASS Utility Includes
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gemm_complex.h"

///////////////////////////////////////////////////////////////////////////////////////////////////

// Define the overal warp-level problem shape
int const kM = 27;
int const kN = 31;
int const kK = 17;

///////////////////////////////////////////////////////////////////////////////////////////////////

// Define a warp-level GEMM operator.
//
// This template could be part of the CUTLASS Template Library or implemented internally. This
// wraps the matrix multiply operation and epilogue with a GEMM-like interface that can be
// instantiated in device code.

namespace cutlass {
namespace gemm {
namespace warp {

template <
  typename Shape,
  typename InstructionShape,
  typename ElementA,
  typename LayoutA,
  typename ElementB,
  typename LayoutB,
  typename ElementC,
  typename LayoutC,
  typename ElementScalar
>
class GemmTensorOp {
public:

  using WarpShape = GemmShape<
    ((Shape::kM + InstructionShape::kM - 1) / InstructionShape::kM) * InstructionShape::kM,
    ((Shape::kN + InstructionShape::kN - 1) / InstructionShape::kN) * InstructionShape::kN,
    InstructionShape::kK
  >;

  using MmaWarp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
    WarpShape,
    InstructionShape,
    double,                             // Data type of A elements
    cutlass::layout::RowMajor,          // Layout of A matrix
    double,                             // Data type of B elements
    cutlass::layout::ColumnMajor,       // Layout of B matrix
    double,                             // Data type of C elements
    cutlass::layout::RowMajor           // Layout of C matrix
  >::Type;
 
  // Number of 'K groups' 
  int const kKgroups = (Shape::kK + InstructionShape::kK - 1) / InstructionShape::kK;

  // Define a 'FragmentIterator' to iterate over slices of accumulators
  using FragmentIterator = cutlass::epilogue::warp::FragmentIteratorTensorOp<
    typename MmaWarp::Shape,
    InstructionShape,
    double,
    typename MmaWarp::Policy::Operator::FragmentC,
    cutlass::layout::RowMajor
  >;

  // Define an epilogue 'Tile Iteterator' to iterate over slices of elements in Shared Memory
  using AccumulatorTileIterator = cutlass::epilogue::warp::TileIteratorTensorOpCanonical<
    typename MmaWarp::Shape,
    InstructionShape,
    double,
    cutlass::layout::RowMajor
  >;

  using TensorRefA = typename MmaWarp::IteratorA::TensorRef;
  using TensorRefB = typename MmaWarp::IteratorB::TensorRef;
  using TensorRefC = typename AccumulatorTileIterator::TensorRef;

public:
  CUTLASS_HOST_DEVICE
  GemmTensorOp() { }

  CUTLASS_DEVICE
  void operator()(
    ElementScalar alpha, 
    TensorRefA ref_A, 
    TensorRefB ref_B, 
    ElementScalar beta,
    TensorRefC ref_C,
    TensorRefC ref_D,
    int lane_id) const {
  
    // Instantiate iterators pointing to slices of the A and B matrices in shared memory
    typename MmaWarp::IteratorA iter_A(ref_A, {Shape::kM, Shape::kK}, lane_id);
    typename MmaWarp::IteratorB iter_B(ref_B, {Shape::kK, Shape::kN}, lane_id);

    // Instantiate and clear accumulator tile holding the C matrix
    typename MmaWarp::FragmentC accum;
    accum.clear();
  
    // Instantiate the warp-level matrix multiply operator
    MmaWarp mma_op;

    // Instantiate fragments holding the slice of the matrix held by each warp
    typename MmaWarp::FragmentA frag_A[2];
    typename MmaWarp::FragmentB frag_B[2];
      
    // Load fragments from shared memory
    iter_A.load(frag_A[0]);
    iter_B.load(frag_B[0]);

    ++iter_A;
    ++iter_B;

    // Load fragments from shared memory
    CUTLASS_PRAGMA_UNROLL
    for (int k = 0; k < kKgroups; ++k) {

      // Load fragments from shared memory
      iter_A.load(frag_A[(k + 1) % 2]);
      iter_B.load(frag_B[(k + 1) % 2]);

      ++iter_A;
      ++iter_B;

      // Compute the matrix multiply
      mma_op(accum, frag_A[k % 2], frag_B[k % 2], accum);
    }
  
    // Instantiate iterators
    FragmentIterator accum_frag_it(accum);
    AccumulatorTileIterator source_tile_it(ref_C, {Shape::kM, Shape::kN}, lane_id);
    AccumulatorTileIterator dest_tile_it(ref_D, {Shape::kM, Shape::kN}, lane_id);

    // Define function objects for linear scaling operation
    cutlass::multiplies<typename FragmentIterator::Fragment> mul_source;
    cutlass::multiply_add<typename FragmentIterator::Fragment> mul_add_accumulator;

    // Iterate over the epilogue components
    CUTLASS_PRAGMA_UNROLL
    for (int idx = 0; idx < FragmentIterator::kIterations; ++idx) {

      // Define storage for slices of the accumulators
      typename FragmentIterator::Fragment accum_fragment;
      typename FragmentIterator::Fragment source_fragment;

      // Select a slice of accumulators from the accumulator tile
      accum_frag_it.load(accum_fragment);
      ++accum_frag_it;

      // Load a corresponding slice from Shared memory
      source_tile_it.load(source_fragment);
      ++source_tile_it;

      // Compute linear scaling - alpha * AB + beta * C
      source_fragment = mul_source(beta, source_fragment);
      accum_fragment = mul_add_accumulator(alpha, accum_fragment, source_fragment);

      // Store the result to shared memory
      dest_tile_it.store(accum_fragment);
      ++dest_tile_it;
    }
  }
};

} // namespace warp
} // namespace gemm
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////////////////////////

// Sample kernel demonstrating a collective GEMM operation by a warp on arbitrary matrices held
// in Shared Memory.
__global__ void kernel(
  double *D_gmem, 
  double alpha, 
  double const *A_gmem, 
  double const *B_gmem, 
  double beta,
  double const *C_gmem) {

  // Define several matrices in shared memory
  __shared__ double A[kM][kK];
  __shared__ double B[kN][kK];
  __shared__ double C[kM][kN];

  // Copy data into SMEM
  if (threadIdx.x == 0) {
    CUTLASS_PRAGMA_NO_UNROLL
    for (int m = 0; m < kM; ++m) {
      for (int k = 0; k < kK; ++k) {
        A[m][k] = A_gmem[m * kK + k];
      }
    }
    CUTLASS_PRAGMA_NO_UNROLL
    for (int n = 0; n < kN; ++n) {
      for (int k = 0; k < kK; ++k) {
        B[n][k] = B_gmem[n * kK + k];
      }
    }
    CUTLASS_PRAGMA_NO_UNROLL
    for (int m = 0; m < kM; ++m) {
      CUTLASS_PRAGMA_NO_UNROLL
      for (int n = 0; n < kN; ++n) {
        C[m][n] = C_gmem[m * kN + n];
      }
    }
  }

  __syncthreads();
  
  //
  // Instantiate a warp-level matrix multiply operator given the fundamental instruction shape (8x8x4),
  // overall shape, data type of each operand, and layout of each operand.
  //

  using GemmTensorOp = cutlass::gemm::warp::GemmTensorOp<
    cutlass::gemm::GemmShape<kM, kN, kK>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    double,                             // Data type of A elements
    cutlass::layout::RowMajor,          // Layout of A matrix
    double,                             // Data type of B elements
    cutlass::layout::ColumnMajor,       // Layout of B matrix
    double,                             // Data type of C elements
    cutlass::layout::RowMajor,          // Layout of C matrix
    double                              // Scalar type of alpha and beta
  >;

  // Instantiate the GEMM operator
  GemmTensorOp gemm;

  // Execute the warp-level GEMM operation
  gemm(
    alpha, 
    {&A[0][0], kK},
    {&B[0][0], kK},
    beta,
    {&C[0][0], kN},
    {&C[0][0], kN},
    threadIdx.x);

  __syncthreads();
  
  // Copy data into SMEM
  if (threadIdx.x == 0) {
    CUTLASS_PRAGMA_NO_UNROLL
    for (int m = 0; m < kM; ++m) {
      CUTLASS_PRAGMA_NO_UNROLL
      for (int n = 0; n < kN; ++n) {
        D_gmem[m * kN + n] = C[m][n];
      }
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to canonical warp-level GEMM operation
int main(int argc, const char *arg[]) {

  bool notSupported = false;

  // CUTLASS must be compiled with CUDA 11 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "NVIDIA Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!((props.major * 10 + props.minor) >= 80)) {
    std::cerr << "This example requires compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    // Return 0 so tests are considered passing if run on unsupported platforms.
    return 0;
  }

  cutlass::HostTensor<double, cutlass::layout::RowMajor> A({kM, kK});
  cutlass::HostTensor<double, cutlass::layout::ColumnMajor> B({kK, kN});
  cutlass::HostTensor<double, cutlass::layout::RowMajor> C({kM, kN});
  cutlass::HostTensor<double, cutlass::layout::RowMajor> D({kM, kN});

  uint64_t seed = 2020;
  double max = 8;
  double min = -8;

  cutlass::reference::host::TensorFillRandomUniform(
    A.host_view(),
    seed,
    max,
    min,
    0
  );

  cutlass::reference::host::TensorFillRandomUniform(
    B.host_view(),
    seed + 17,
    max,
    min,
    0
  );

  cutlass::reference::host::TensorFillRandomUniform(
    C.host_view(),
    seed + 31,
    max,
    min,
    0
  );

  A.sync_device();
  B.sync_device();
  C.sync_device();
  D.sync_device();

  dim3 grid(1,1);
  dim3 block(32, 1, 1);

  double alpha = 2.25;
  double beta = 1.24;

  kernel<<< grid, block >>>(
    D.device_data(),
    alpha,
    A.device_data(),
    B.device_data(),
    beta,
    C.device_data()
  );

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    std::cerr << "Failed to synchronize device after kernel launch." << std::endl;
    return -1;
  }

  D.sync_host();
  
  // Compute reference on host
  cutlass::HostTensor<double, cutlass::layout::RowMajor> D_ref({kM, kN}, false);

  cutlass::reference::host::GemmComplex(
    {kM, kN, kK},
    alpha,
    A.host_ref(),
    cutlass::ComplexTransform::kNone,
    B.host_ref(),
    cutlass::ComplexTransform::kNone,
    beta,
    C.host_ref(),
    D_ref.host_ref(),
    double()
  );

  // Verify reference matches computed
  if (!cutlass::reference::host::TensorEquals(
    D.host_view(),
    D_ref.host_view())) {

    std::cerr 
      << "A =\n" << A.host_view() 
      << "\n\nB = \n" << B.host_view() 
      << "\n\nC = " << C.host_view() 
      << "\n\nRef =\n" << D_ref.host_view()
      << "\n\nD =\n" << D.host_view() << "\n\n";

    std::cerr << "Error - device results mismatch host reference." << std::endl;

    return -1;
  }

  std::cout << "Passed" << std::endl;

  return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
