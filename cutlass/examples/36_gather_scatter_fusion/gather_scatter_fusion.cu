#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

// This example fuses gather before GEMM and scatter after GEMM into the same
// GEMM kernel.  Gather and scatter operation is controled by an index vector
// to select rows or columns from A, B, C or D matrices.
//
// Suppose, all matrices are column major.  The pseudo code of the fused kernel
// in this example is essentially
//
//    for (int i = 0; i < problem_size.m(); ++i) {
//      for (int j = 0; j < options.index_size; ++j) {
//        int b_c_d_col = tensor_indices.at({j, 0});
//
//        for (int k = 0; k < options.index_size; ++k) {
//            int a_col = tensor_indices.at({k, 0});
//            tensor_d_ref.at({i, b_c_d_col}) +=
//              alpha * tensor_a.at({i, a_col}) * tensor_b.at({k, b_c_d_col});
//        }
//      }
//
// Note that the index vector contains unique random integers with max to be N - 1
//
// The gather/scatter operation works best when we can still keep the biggest
// alignment. For example, when the matrix is row major, we select rows. When
// the matrix is column major, we selct columns.
//
// Not all the combination of gather and scatter are legal. For example, if A is
// row major and C/D is column major, we cannot gather A and scatter C/D at the
// same time.
//
// Also, we don't check the index value is legal and index array point is valid
// for the sake of the performance.
 
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <random>
#include <numeric>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"
#include "helper.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  cutlass::gemm::GemmCoord problem_size;
  int index_size;

  bool reference_check;
  int iterations;
  
  Options():
    help(false),
    problem_size({248, 1024, 1024}),
    index_size(240),
    reference_check(true),
    iterations(20) { }

  bool valid() {
    return true;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());

    cmd.get_cmd_line_argument("index_size", index_size);
    
    cmd.get_cmd_line_argument("iterations", iterations);

  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "36_gather_scatter_fusion example\n\n"
      << "  This example uses the CUTLASS Library to fuse gather/scatter into GEMM\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --index_size=<int>          size of N dimension index\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/36_gather_scatter_fusion/36_gather_scatter_fusion --m=1024 --n=512 --k=1024 \\\n"
      << "     --index_size=128\n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = problem_size.product();
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = float;                   // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = cutlass::half_t;;             // <- data type of elements in input matrix A
using ElementInputB = cutlass::half_t;;             // <- data type of elements in input matrix B
using ElementOutput = float;                        // <- data type of elements in output matrix D

// The code section below describes matrix layout of input and output matrices.
// Column Major for Matrix A, B and C.
//
using LayoutInputA = cutlass::layout::ColumnMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::ColumnMajor;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock =
    cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32 
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 16>;  // <- MMA Op tile M = 8, N = 8, K = 4
// 16, 8, 8 -> Turing
// 16, 8, 16 -> Ampere

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??

// Define the epilogue operation as LinearCombination. This is approximately equal to
//
//    d_ij = alpha * sum_k(a_ik * b_kj) + c_ij
//
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        // <- data type of output matrix
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // <- this is the number of elements per
                                                          // vectorized memory access. For half
                                                          // precision, it's 8 elements. This becomes
                                                          // the vector width of math instructions in
                                                          // epilogue too
    ElementAccumulator,                                   // <- data type of accumulator
    ElementComputeEpilogue>;                              // <- data type for alpha in linear combination function

// Number of pipelines you want to use
constexpr int NumStages = 5;
// Ampere -> 4/5
// Turing -> 2

using Gemm = cutlass::gemm::device::GemmUniversal<ElementInputA,
                                                  LayoutInputA,
                                                  ElementInputB,
                                                  LayoutInputB,
                                                  ElementOutput,
                                                  LayoutOutput,
                                                  ElementAccumulator,
                                                  MMAOp,
                                                  SmArch,
                                                  ShapeMMAThreadBlock,
                                                  ShapeMMAWarp,
                                                  ShapeMMAOp,
                                                  EpilogueOp,
                                                  SwizzleThreadBlock,
                                                  NumStages,
                                                  8,     /*alignmentA*/
                                                  8,     /*alignmengB*/
                                                  cutlass::arch::OpMultiplyAdd,
                                                  cutlass::ComplexTransform::kNone,
                                                  cutlass::ComplexTransform::kNone,
                                                  true,  /*GatherA*/
                                                  true,  /*GatherB*/
                                                  true   /*ScatterD*/
                                                 >;

int run(Options &options) {

  // ================================================================================
  // Initialization setup

  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size = options.problem_size;

  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size_real(problem_size.m(),
                                             options.index_size,
                                             options.index_size);

  // Initialize tensors using CUTLASS helper functions
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(
      problem_size.mk());  // <- Create matrix A with dimensions M x K
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(
      cutlass::make_Coord(options.index_size, problem_size.n()));  // <- Create matrix B with dimensions K x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(
      problem_size.mn());  // <- Create matrix C with dimensions M x N 
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d_scattered(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // CUTLASS kernel

  // Fill input and output matrices on host using CUTLASS helper functions
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8),
      0);  // <- Fill matrix A on host with uniform-distribution random data

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8),
      0);  // <- Fill matrix B on host with uniform-distribution random data

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      1,
      ElementOutput(7),
      ElementOutput(-8),
      0);  // <- Fill matrix C on host with uniform-distribution random data

  cutlass::reference::host::TensorFill(
    tensor_d_scattered.host_view());  // <- fill matrix D on host with zeros

  cutlass::HostTensor<int, LayoutOutput> tensor_indices(
      {options.index_size, 1});  // <- Create scatter indices with dimensions val_len x 1

  // <- Fill tensor_b_indices on host with unique random integers
  std::vector<int> to_fill(problem_size.n()) ; // vector with ints.
  std::iota (std::begin(to_fill), std::end(to_fill), 0); // Fill with 0, 1, ...., problem_size.n()
  std::random_shuffle(to_fill.begin(), to_fill.end());
  memcpy(tensor_indices.host_data(), to_fill.data(), options.index_size * sizeof(int));

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_indices.sync_device();
  tensor_c.sync_device();
  tensor_d_scattered.sync_device();

  // Initialize alpha/beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(1);

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm, 
      problem_size_real,                  // <- problem size of matrix multiplication
      split_k_slices,                     // <- k-dimension split factor
      {alpha, beta},                      // <- alpha, beta
      tensor_a.device_data(),             // <- reference to matrix A on device
      tensor_b.device_data(),             // <- reference to matrix B on device
      tensor_c.device_data(),             // <- reference to matrix C on device
      tensor_d_scattered.device_data(),   // <- reference to matrix D on device
      tensor_a.layout().capacity(problem_size.mk()),
      tensor_b.layout().capacity(cutlass::make_Coord(options.index_size, problem_size.n())),
      tensor_c.layout().capacity(problem_size.mn()),
      tensor_d_scattered.layout().capacity(problem_size.mn()),
      tensor_a.layout().stride(),
      tensor_b.layout().stride(),
      tensor_c.layout().stride(),
      tensor_d_scattered.layout().stride(),
      tensor_indices.device_data(),       // <- pointer to index vector to gather A on device
      tensor_indices.device_data(),       // <- pointer to index vector to gather B on device
      tensor_indices.device_data()};      // <- pointer to index vector to scatter D on device

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm_op;

  // Check the problem size is supported or not 
  cutlass::Status status = gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(status);

  // CPU reference calculation
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d_ref(problem_size.mn());
  cutlass::reference::host::TensorFill(
    tensor_d_ref.host_view());  // <- Fill matrix D on host with zeros

  status = gemm_op();
  hipDeviceSynchronize();
  CUTLASS_CHECK(status);

  if (options.reference_check) {
    for (int i = 0; i < problem_size.m(); ++i) {
      for (int j = 0; j < options.index_size; ++j) {
        int b_c_d_col = tensor_indices.at({j, 0});

        for (int k = 0; k < options.index_size; ++k) {
            int a_col = tensor_indices.at({k, 0});
            tensor_d_ref.at({i, b_c_d_col}) +=
              alpha * tensor_a.at({i, a_col}) * tensor_b.at({k, b_c_d_col});
        }
       
        tensor_d_ref.at({i, b_c_d_col}) += (beta * tensor_c.at({i, b_c_d_col}));
      }
    }

    // Copy output data from CUTLASS and reference kernel to host for comparison
    tensor_d_scattered.sync_host();
  
    bool passed = cutlass::reference::host::TensorEquals(
                    tensor_d_scattered.host_view(),
                    tensor_d_ref.host_view());

    if (!passed) {
      std::cout << "Failed!\n";

      std::stringstream fname;
      fname << "error_gather_GEMM_scatter_fusion.txt";
      std::cerr << "Dumping results in " << fname.str() << "\n";

      std::ofstream file(fname.str());

      file 
        << "A =\n" << tensor_a.host_view()
        << "\nB =\n" << tensor_b.host_view()
        << "\nindices =\n" << tensor_indices.host_view()
        << "\nC =\n" << tensor_c.host_view()
        << "\n\nReference =\n" << tensor_d_ref.host_view()
        << "\nComputed =\n" << tensor_d_scattered.host_view();
      return -1;
    } else {
      std::cout << "Passed!\n";
    }
  }

  // Result structure
  Result result;

  //
  // Construct events
  //

  hipEvent_t events[2];

  for (auto & event : events) {
    result.error = hipEventCreate(&event);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
      return -1;
    }
  }

  // Record an event at the start of a series of GEMMs
  result.error = hipEventRecord(events[0]);
  if (result.error != hipSuccess) {
    std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
    return -1;
  }

  //
  // Run profiling loop
  //

  for (int iter = 0; iter < options.iterations; ++iter) {
    // Launch initialized CUTLASS kernel
    status = gemm_op();
    CUTLASS_CHECK(status);
  }

  //
  // Stop profiling loop
  //

  // Record an event when the GEMMs are complete
  result.error = hipEventRecord(events[1]);
  if (result.error != hipSuccess) {
    std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
    return -1;
  }

  // Wait for work on the device to complete.
  result.error = hipEventSynchronize(events[1]);
  if (result.error != hipSuccess) {
    std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
    return -1;
  }

  // Measure elapsed runtime
  float runtime_ms = 0;
  result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  if (result.error != hipSuccess) {
    std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
    return -1;
  }

  // Compute average runtime and GFLOPs.
  result.runtime_ms = double(runtime_ms) / double(options.iterations);
  result.gflops = options.gflops(result.runtime_ms / 1000.0);

  // Cleanup
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }

  std::cout << "Runtime: " << result.runtime_ms << " ms\n";
  std::cout << " GFLOPs: " << result.gflops << "\n";

  return 0;
}

int main(int argc, const char ** argv) {
  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major > 8 || (props.major == 8 && props.minor >= 0))) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << "\n";
    return 0;
  }

  if (!options.valid()) {
    std::cerr << "Invalid problem." << "\n";
    return -1;
  }

  return run(options);
}
