#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**

*/

#include <cmath>
#include <iostream>
#include <vector>
#include <limits>

#include "cutlass/cutlass.h"
#include "cutlass/arch/memory.h"
#include "cutlass/arch/memory_sm75.h"
#include "cutlass/gemm/device/gemm_complex.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"

#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_reduce.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/error_metrics.h"
#include "cutlass/util/tensor_view_io.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

#include "gemm_with_softmax.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

#define TRACE(x) { std::cout << "gemm_softmax.cu:" << __LINE__ << "  " << x << std::endl; }

/////////////////////////////////////////////////////////////////////////////////////////////////

enum class Disposition {
  kPassed,
  kIncorrect,
  kNotVerified
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::gemm::GemmCoord problem_size;
  int batch_count;
  int iterations;
  unsigned seed;
  float alpha;
  float beta;
  bool verification_enabled;
  double tolerance;

  Options():
    help(false),
    problem_size({16, 24, 64}),
    batch_count(1),             // As a temporary limitation to the test bench, batch count must be 1. The kernels support arbitrary batching.
    iterations(20),
    seed(2022),
    alpha(1),
    beta(),
    verification_enabled(true),
    tolerance(0.01)
  { }

  bool valid() {

    return true;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);

    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("verify", verification_enabled);
    cmd.get_cmd_line_argument("seed", seed);
    cmd.get_cmd_line_argument("tolerance", tolerance);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "35_gemm_softmax example\n\n"
      << "  This example uses the CUTLASS Library to compute GEMM + Softmax for arbitrary problem sizes.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --seed=<int>                Random number seed (1*)\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform (0 to disable profiling).\n\n"
      << "  --verify=<bool>             If true, performs reference calculation.\n\n"
      << "  --tolerance <float>         Error tolerance\n"
    ;

    out << "\n\nExamples:\n\n"
      << "$ ./examples/35_gemm_softmax/35_gemm_softmax --m=1024 --n=512 \\\n"
      << "     --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Returns true if the environment and Toolkit support this
  bool supported(bool verbose = true) const {

    // Ampere Tensor Core operations exposed with mma.sync and ldmatrix are first available
    // in CUDA 11.0.
    //
    // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
    if (!(__CUDACC_VER_MAJOR__ >= 11)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
      }
      return false;
    }

    hipDeviceProp_t props;

    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
      if (verbose) {
        std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
      }
      return false;
    }

    if (!((props.major * 10 + props.minor) >= 80)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
                  << std::endl;
      }
      return false;
    }

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Testbed {

  //
  // Type definitions
  //


  using ElementA = cutlass::half_t;
  using ElementB = cutlass::half_t;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::half_t;
  using ElementCompute = float;
  using ElementSoftmax = cutlass::half_t;

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;

  using GemmSoftmax = cutlass::GemmSoftmax<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC,
    ElementCompute
  >;

  using ElementN = typename GemmSoftmax::ElementN;
  using LayoutC = typename GemmSoftmax::LayoutC;

  //
  // Data members
  //

  Options const &options;

  cutlass::HostTensor<ElementA, LayoutA>        tensor_A;
  cutlass::HostTensor<ElementB, LayoutB>        tensor_B;
  cutlass::HostTensor<ElementC, LayoutC>        tensor_C;
  cutlass::HostTensor<ElementD, LayoutC>        tensor_D;
  cutlass::HostTensor<ElementN, LayoutC>        tensor_N;
  cutlass::HostTensor<ElementSoftmax, LayoutC>  tensor_Softmax;

  cutlass::HostTensor<ElementD, LayoutC>        reference_D;
  cutlass::HostTensor<ElementN, LayoutC>        reference_N;
  cutlass::HostTensor<ElementSoftmax, LayoutC>  reference_Softmax;

  //
  // Methods
  //

  Testbed(
    Options const &options_
  ):
    options(options_)
  {

    tensor_A.reset({options.problem_size.m(), options.problem_size.k()});
    tensor_B.reset({options.problem_size.k(), options.problem_size.n()});

    tensor_C.reset({options.problem_size.m(), options.problem_size.n()});
    tensor_D.reset({options.problem_size.m(), options.problem_size.n()});

    tensor_N.reset({options.problem_size.m(), 1});
    tensor_Softmax.reset({options.problem_size.m(), options.problem_size.n()});

    reference_D.reset({options.problem_size.m(), options.problem_size.n()}, false);
    reference_N.reset({options.problem_size.m(), 1}, false);
    reference_Softmax.reset({options.problem_size.m(), options.problem_size.n()}, false);
  }

  /// Run
  Disposition run() {

    Disposition disposition = Disposition::kNotVerified;

    //
    // Initialize the workspace
    //

    initialize();

    //
    // Launch device kernel
    //
    cutlass::Status status = cutlass::Status::kSuccess;

    status = execute_device_kernel();

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Device execution failed." << std::endl;
      return disposition;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Device synchronize failed with error "
        << hipGetErrorString(result) << std::endl;
      return disposition;
    }

    //
    // Compute the reference
    //
    compute_reference();

    //
    // Verify
    //

    if (options.verification_enabled) {

      bool passed = verify();

      if (passed) {
        disposition = Disposition::kPassed;
      }
      else {
        disposition = Disposition::kIncorrect;
      }
    }

    //
    // Profiling
    //
    if (options.iterations) {
      profile();
    }

    return disposition;
  }

  /// Random initialization
  void initialize() {

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_A.host_view(),
        options.seed,
        ElementD(5),
        ElementD(-5),
        0
      );

    cutlass::reference::host::TensorFillRandomUniform(
      tensor_B.host_view(),
        options.seed + 19,
        ElementD(5),
        ElementD(-5),
        0
      );

    cutlass::reference::host::TensorFill(
      reference_D.host_view(),
      ElementD()
      );

    cutlass::reference::host::TensorFill(
      reference_N.host_view(),
      ElementN()
    );

    cutlass::reference::host::TensorFill(
      reference_Softmax.host_view(),
      ElementSoftmax()
    );

    tensor_A.sync_device();
    tensor_B.sync_device();
    tensor_D.sync_device();
    tensor_N.sync_device();
    tensor_Softmax.sync_device();
  }

  cutlass::Status execute_device_kernel() {

    cutlass::Status status = cutlass::Status::kSuccess;

    //
    // Setup arguments
    //

    GemmSoftmax::Arguments args(
      options.problem_size,
      options.batch_count,
      tensor_A.device_ref(),
      tensor_B.device_ref(),
      tensor_C.device_ref(),
      tensor_D.device_ref(),
      {
        ElementCompute(options.alpha),
        ElementCompute(options.beta)
      },
      tensor_N.device_ref(),
      tensor_Softmax.device_ref()
    );

    //
    // Launch
    //

    GemmSoftmax gemm_softmax;

    // Initialize
    status = gemm_softmax.initialize(args);
    if (status != cutlass::Status::kSuccess) {
      return status;
    }

    // Run
    status = gemm_softmax();

    return status;
  }

  /// Reference calculation
  void compute_reference() {

    // Compute GEMM

    cutlass::reference::host::GemmComplex(
      options.problem_size,
      options.alpha,
      tensor_A.host_ref(),
      cutlass::ComplexTransform::kNone,
      tensor_B.host_ref(),
      cutlass::ComplexTransform::kNone,
      options.beta,
      tensor_C.host_ref(),
      reference_D.host_ref(),
      double()
    );

    // Compute the norm
    for (int m = 0; m < options.problem_size.m(); ++m) {
      reference_N.at({m, 0}) = reference_D.at({m, 0});
      for (int n = 1; n < options.problem_size.n(); ++n) {
        reference_N.at({m, 0}) = std::max(reference_N.at({m, 0}), ElementN(reference_D.at({m, n})));
      }
    }

    // Compute softmax
    for (int m = 0; m < options.problem_size.m(); ++m) {

      float sum = float();

      for (int n = 0; n < options.problem_size.n(); ++n) {
        sum += std::exp( float(reference_D.at({m, n})) - float(reference_N.at({m, 0})) );
      }

      float inv_sum = float(1.0f / sum);

      for (int n = 0; n < options.problem_size.n(); ++n) {

        reference_Softmax.at({m, n}) = ElementSoftmax(
          std::exp( float(reference_D.at({m, n})) - float(reference_N.at({m, 0})) ) * inv_sum
        );
      }
    }
  }

  /// Emits all tensor values
  void emit_results() {
    std::cout << "D = \n" << tensor_D.host_view() << "\n\n";
    std::cout << "N = \n" << tensor_N.host_view() << "\n\n";
    std::cout << "Softmax = \n" << tensor_Softmax.host_view() << "\n\n";
    std::cout << "Reference N = \n" << reference_N.host_view() << "\n\n";
    std::cout << "Reference D = \n" << reference_D.host_view() << "\n\n";
    std::cout << "Reference Softmax = \n" << reference_Softmax.host_view() << "\n\n";
  }

  /// Verifies the reference matches
  bool verify() {

    tensor_D.sync_host();
    tensor_N.sync_host();
    tensor_Softmax.sync_host();

    double const kThreshold = options.tolerance;

    // Verification checks - set any of these to 'true' to override the verification checks.
    bool verified_D = false;
    bool verified_N = false;
    bool verified_Softmax = false;

    // Verify softmax output
    if (!verified_D) {

      double norm_diff = cutlass::reference::host::TensorNormDiff(
        tensor_D.host_view(),
        reference_D.host_view());

      double norm_reference = cutlass::reference::host::TensorNorm(
        reference_D.host_view());

      double rel_error = norm_diff / norm_reference;

      if (rel_error > kThreshold) {
        std::cerr << "\n\nTensor D Relative error: " << rel_error << std::endl;
      }
      else {
        verified_D = true;
      }
    }

    if (!verified_N) {

      double norm_diff = cutlass::reference::host::TensorNormDiff(
        tensor_N.host_view(),
        reference_N.host_view());

      double norm_reference = cutlass::reference::host::TensorNorm(
        reference_N.host_view());

      double rel_error = norm_diff / norm_reference;

      if (rel_error > kThreshold) {
        std::cerr << "\n\nTensor N Relative error: " << rel_error << std::endl;
      }
      else {
        verified_N = true;
      }
    }

    if (!verified_Softmax) {

      double norm_diff = cutlass::reference::host::TensorNormDiff(
        tensor_Softmax.host_view(),
        reference_Softmax.host_view());

      double norm_reference = cutlass::reference::host::TensorNorm(
        reference_Softmax.host_view());

      double rel_error = norm_diff / norm_reference;

      if (rel_error > kThreshold) {
        std::cerr << "\n\nSoftmax Relative error: " << rel_error << std::endl;
      }
      else {
        verified_Softmax = true;
      }
    }

    if (!verified_D || !verified_N || !verified_Softmax) {

      std::cerr << "Verification check failed for tensor Softmax" << std::endl;

      emit_results();

      // Summarize which checks failed
      if (!verified_D) {
        std::cerr << "Verification of D tensor failed\n";
      }

      if (!verified_N) {
        std::cerr << "Verification of N tensor failed\n";
      }

      if (!verified_Softmax) {
        std::cerr << "Verification of Softmax tensor failed\n";
      }

      return false;
    }

    return true;
  }

  /// Profiles
  bool profile() {

    //
    // Profile
    //

    cutlass::Status status = cutlass::Status::kSuccess;
    hipError_t result;
    hipEvent_t events[2];
    int const kIterations = options.iterations;

    for (hipEvent_t &evt : events) {
      result = hipEventCreate(&evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[0]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (int iter = 0; iter < kIterations; ++iter) {

      status = execute_device_kernel();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Device execution failed." << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "hipDeviceSynchronize() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    float elapsed_ms = 0;
    result = hipEventElapsedTime(&elapsed_ms, events[0], events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventElapsedTime() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (hipEvent_t &evt : events) {
      result = hipEventDestroy(evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventDestroy() failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    int64_t flops = int64_t(options.problem_size.m()) * options.problem_size.n() * options.problem_size.k() * 2;
    int64_t bytes = (sizeof(ElementD) * 2 + sizeof(ElementSoftmax)) * options.problem_size.m() * options.problem_size.n();

    double gflops_per_second = double(flops) * kIterations / double(elapsed_ms / 1000.0f) / double(1.0e9);
    double gbytes_per_second = double(bytes) * kIterations / double(elapsed_ms / 1000.0f) / double(1 << 30);

    std::cout << "         Problem: "
              << options.problem_size.m() << "-by-" << options.problem_size.n() << "-by-" << options.problem_size.k()
              << std::endl;

    std::cout << "         Runtime: " << elapsed_ms << " ms\n" << std::endl;

    std::cout << "          GFLOPs: " << gflops_per_second << "  GFLOPs" << std::endl;
    std::cout << "Memory bandwidth: " << gbytes_per_second << "  GiB/s" << std::endl;

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv) {

  // Options parsing
  Options options;
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (!options.supported()) {
    return 0;
  }

  // Run
  Testbed testbed(options);

  Disposition disposition = testbed.run();

  std::cout << std::endl;

  switch (disposition) {
    case Disposition::kPassed:
      std::cout << "Passed" << std::endl;
      break;
    case Disposition::kIncorrect:
      std::cout << "Incorrect" << std::endl;
      break;
    case Disposition::kNotVerified:
      std::cout << "Not verified" << std::endl;
      break;
  }

  return (disposition == Disposition::kPassed ? 0 : -1);
}


/////////////////////////////////////////////////////////////////////////////////////////////////

