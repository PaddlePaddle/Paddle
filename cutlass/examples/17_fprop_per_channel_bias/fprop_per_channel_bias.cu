#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
The convolution version of 12_gemm_bias_relu.  Similarly, we put bias vector in Operand C and the
rest is the same as normal convolution.
*/

#include <iostream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/host_reorder.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

// The code section below describes datatype for input, output tensors and computation between
// elements 
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = ElementAccumulator; // Data type of epilogue computation
using ElementInputA = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;             // Data type of elements in input tensor
using ElementOutput = float;                       // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;  // Threadblock tile shape

// This code section describes tile size a warp will compute
using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;         // Warp tile shape

// This code section describes the size of MMA op
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;    // TensorCore instruction shape

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipelines you want to use
constexpr int NumStages = 4;

// This code section describe iterator algorithm selected is Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// This code section describes the epilogue part of the kernel, we use default value
using EpilogueOp = cutlass::epilogue::thread::LinearCombinationRelu<
    ElementOutput,                                        // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // The number of elements per vectorized.
                                                          // memory access. This becomes the vector width of
                                                          // math instructions in the epilogue too.
    ElementAccumulator,                                   // Data type of accumulator
    ElementComputeEpilogue,                               // Data type for alpha in linear combination
    cutlass::epilogue::thread::ScaleType::NoBetaScaling>; // alpha X C + per channel bias


using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  IteratorAlgorithm
>::Kernel;

using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

int run() {

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(      
    {1, 7, 7, 512},                               // activation 
    {512, 3, 3, 512},                             // filter
    {1, 1, 1, 1},                                 // padding
    {1, 1},                                       // striding
    {1, 1},                                       // dilation
    cutlass::conv::Mode::kCrossCorrelation,       // mode (convolution or cross-correlation)
    1                                             // split-k slices
  );

  // Initialize tensors using CUTLASS helper functions
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(problem_size.activation_extent());
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(problem_size.filter_extent());

  // Create tensor C with dimensions 1x1x1xk which is the bias vector
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c_bias({1, 1, 1, problem_size.K});

  // Create tensor D used to store output from CUTLASS kernel
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(problem_size.output_extent());
  // Create matrix D with dimensions M x N used to store output from reference
  // kernel
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(problem_size.output_extent());

  // Fill input and output matrices on host using CUTLASS helper functions
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(4),
      ElementInputA(-4),
      0);  // <- Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(4),
      ElementInputB(-4),
      0);  // <- Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c_bias.host_view(),
      1,
      ElementOutput(4),
      ElementOutput(-4),
      0);  // <- Fill matrix C on host with uniform-distribution random data
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- fill matrix D on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- fill matrix D for reference on host with zeros

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c_bias.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  // Initialize alpha for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),              // <- reference to tensor A on device
    tensor_b.device_ref(),              // <- reference to tensor B on device
    // tensor C  is treated as the bias vector. We can enable the CONV
    // to project away the N, H, W dimension by setting the stride to zero.
    {tensor_c_bias.device_data(), LayoutOutput::Stride(0)},
    tensor_d.device_ref(),              // <- reference to tensor D on device
    {alpha} };                    

  // Instantiate CUTLASS kernel depending on templates
  ImplicitGemm implicit_gemm_op;

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  cutlass::Status status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(status);

  // Launch initialized CUTLASS kernel
  status = implicit_gemm_op();

  CUTLASS_CHECK(status);

  //
  // Create instantiation for device reference conv kernel
  //

  // Launch device reference to compute strictly the product A * B
  cutlass::reference::device::Conv2d<
      ElementInputA, 
      LayoutInputA, 
      ElementInputB, 
      LayoutInputB, 
      ElementOutput,
      LayoutOutput, 
      ElementComputeEpilogue, 
      ElementAccumulator,
      cutlass::NumericConverter<ElementOutput, ElementComputeEpilogue>>
    (
      cutlass::conv::Operator::kFprop, 
      problem_size, 
      tensor_a.device_ref(),
      tensor_b.device_ref(), 
      tensor_c_bias.device_ref(), 
      tensor_ref_d.device_ref(),
      alpha, ElementComputeEpilogue(0)
    );

  // Wait for kernels to finish
  hipDeviceSynchronize();

  // Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d.sync_host();
  tensor_ref_d.sync_host();

  // Compute bias + relu in host code
  for (int n = 0; n < problem_size.N; ++n) {
    for (int p = 0; p < problem_size.P; ++p) {
      for (int q = 0; q < problem_size.Q; ++q) {
        for (int k = 0; k < problem_size.K; ++k) {
          
          tensor_ref_d.at({n, p, q, k}) =
              std::max(ElementOutput(0),
                       ElementOutput(tensor_ref_d.at({n, p, q, k}) +
                                     tensor_c_bias.at({0, 0, 0, k})));
        }
      }
    }
  }

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  std::cout << (cutlass::reference::host::TensorEquals(tensor_d.host_view(),
                                                       tensor_ref_d.host_view())
                    ? "Passed"
                    : "Failed")
            << std::endl;

  CUTLASS_CHECK(status);
  return 0;
}

int main(int argc, char const **args) {

  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major > 8 || (props.major == 8 && props.minor >= 0))) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  return run();
}

/////////////////////////////////////////////////////////////////////////////////////////////////
