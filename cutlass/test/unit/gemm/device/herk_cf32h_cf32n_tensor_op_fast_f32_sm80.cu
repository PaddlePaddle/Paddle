/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide HERK interface

  
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/blas3.h"
#include "cutlass/gemm/device/rank_k.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/rank_k_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed_rank_k_universal.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
// HERK operator on HIPBLAS_OP_N (column-major) input layouts 
TEST(SM80_Device_Herk_cf32n_cf32n_l_tensor_op_fast_f32, 64x64x16_32x32x16) {

  using ElementA = cutlass::complex<float>;
  using LayoutA = cutlass::layout::ColumnMajor;

  using ElementC = cutlass::complex<float>;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementAccumulator = cutlass::complex<float>;

  using RankK = cutlass::gemm::device::RankK<
    ElementA,
    LayoutA,
    ElementC,
    LayoutC,
    cutlass::FillMode::kLower,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4,     // kStages 
    1,     // AlignmentA
    false, // SplitKSerial
    cutlass::arch::OpMultiplyAddComplexFastF32,
    cutlass::ComplexTransform::kNone,
    cutlass::BlasMode::kHermitian
  >;

  EXPECT_TRUE(test::gemm::device::TestAllRankKUniversal<RankK>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

// HERK operator on HIPBLAS_OP_N (column-major) input layouts 
TEST(SM80_Device_Herk_cf32n_cf32n_u_tensor_op_fast_f32, 64x64x16_32x32x16) {

  using ElementA = cutlass::complex<float>;
  using LayoutA = cutlass::layout::ColumnMajor;

  using ElementC = cutlass::complex<float>;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementAccumulator = cutlass::complex<float>;

  using RankK = cutlass::gemm::device::RankK<
    ElementA,
    LayoutA,
    ElementC,
    LayoutC,
    cutlass::FillMode::kUpper,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4,     // kStages 
    1,     // AlignmentA
    false, // SplitKSerial
    cutlass::arch::OpMultiplyAddComplexFastF32,
    cutlass::ComplexTransform::kNone,
    cutlass::BlasMode::kHermitian
  >;

  EXPECT_TRUE(test::gemm::device::TestAllRankKUniversal<RankK>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////
// HERK operator on HIPBLAS_OP_C (row-major + conj) input layouts
TEST(SM80_Device_Herk_cf32h_cf32n_l_tensor_op_fast_f32, 64x64x16_32x32x16) {

  using ElementA = cutlass::complex<float>;
  using LayoutA = cutlass::layout::RowMajor;

  using ElementC = cutlass::complex<float>;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementAccumulator = cutlass::complex<float>;

  using RankK = cutlass::gemm::device::RankK<
    ElementA,
    LayoutA,
    ElementC,
    LayoutC,
    cutlass::FillMode::kLower,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3,     // kStages 
    1,     // AlignmentA
    false, // SplitKSerial
    cutlass::arch::OpMultiplyAddComplexFastF32,
    cutlass::ComplexTransform::kConjugate,
    cutlass::BlasMode::kHermitian
  >;

  EXPECT_TRUE(test::gemm::device::TestAllRankKUniversal<RankK>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////
// HERK operator on HIPBLAS_OP_C (row-major + conj) input layouts
TEST(SM80_Device_Herk_cf32h_cf32n_u_tensor_op_fast_f32, 64x64x16_32x32x16) {

  using ElementA = cutlass::complex<float>;
  using LayoutA = cutlass::layout::RowMajor;

  using ElementC = cutlass::complex<float>;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementAccumulator = cutlass::complex<float>;

  using RankK = cutlass::gemm::device::RankK<
    ElementA,
    LayoutA,
    ElementC,
    LayoutC,
    cutlass::FillMode::kUpper,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3,     // kStages 
    1,     // AlignmentA
    false, // SplitKSerial
    cutlass::arch::OpMultiplyAddComplexFastF32,
    cutlass::ComplexTransform::kConjugate,
    cutlass::BlasMode::kHermitian
  >;

  EXPECT_TRUE(test::gemm::device::TestAllRankKUniversal<RankK>());
}
/////////////////////////////////////////////////////////////////////////////////////////////////

#endif // #if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
