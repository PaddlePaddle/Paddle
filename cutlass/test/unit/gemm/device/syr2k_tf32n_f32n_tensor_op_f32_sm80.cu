/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide SYRK interface
  
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/blas3.h"
#include "cutlass/gemm/device/rank_2k.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/rank_2k.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed_rank2k_universal.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Syr2k_tf32n_f32n_l_tensor_op_f32, 128x256x32_64x64x32) {

  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementC = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;

  using Rank2K = cutlass::gemm::device::Rank2K<
    ElementA,
    LayoutA,
    ElementB,
    LayoutB,
    ElementC,
    LayoutC,
    cutlass::FillMode::kLower,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllRank2KUniversal<Rank2K>());

}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Syr2k_tf32n_f32n_u_tensor_op_f32, 128x256x32_64x64x32) {

  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementC = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;

  using Rank2K = cutlass::gemm::device::Rank2K<
    ElementA,
    LayoutA,
    ElementB,
    LayoutB,
    ElementC,
    LayoutC,
    cutlass::FillMode::kUpper,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllRank2KUniversal<Rank2K>());

}
/////////////////////////////////////////////////////////////////////////////////////////////////
#endif // #if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
