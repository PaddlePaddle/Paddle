/**************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface
    
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 128x256x1024_64x64x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 256, 1024>,
      cutlass::gemm::GemmShape<64, 64, 1024>,
      cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 256x128x1024_64x64x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 1024>,
      cutlass::gemm::GemmShape<64, 64, 1024>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 128x128x1024_64x64x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 128, 1024>,
      cutlass::gemm::GemmShape<64, 64, 1024>,
      cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 256x64x1024_64x64x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 64, 1024>,
      cutlass::gemm::GemmShape<64, 64, 1024>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 64x256x1024_64x64x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<64, 256, 1024>,
      cutlass::gemm::GemmShape<64, 64, 1024>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 64x128x1024_32x64x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<64, 128, 1024>,
      cutlass::gemm::GemmShape<32, 64, 1024>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 128x64x1024_64x32x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 64, 1024>,
      cutlass::gemm::GemmShape<64, 32, 1024>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 64x64x1024_32x32x1024, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<64, 64, 1024>,
      cutlass::gemm::GemmShape<32, 32, 1024>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 4, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 128x256x512_64x64x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 256, 512>,
      cutlass::gemm::GemmShape<64, 64, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 256x128x512_64x64x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 128, 512>,
      cutlass::gemm::GemmShape<64, 64, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 128x128x512_64x64x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 128, 512>,
      cutlass::gemm::GemmShape<64, 64, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 256x64x512_64x64x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<256, 64, 512>,
      cutlass::gemm::GemmShape<64, 64, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 64x256x512_64x64x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<64, 256, 512>,
      cutlass::gemm::GemmShape<64, 64, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 3, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 64x128x512_32x64x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<64, 128, 512>,
      cutlass::gemm::GemmShape<32, 64, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 4, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 128x64x512_64x32x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<128, 64, 512>,
      cutlass::gemm::GemmShape<64, 32, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 4, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

CUTLASS_TEST_L1(SM80_Device_Gemm_XOR_b1t_b1n_s32t_tensor_op_s32, 64x64x512_32x32x512, {
  using ElementOutput = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;

  using Gemm = cutlass::gemm::device::Gemm<
      cutlass::uint1b_t, cutlass::layout::RowMajor, cutlass::uint1b_t,
      cutlass::layout::ColumnMajor, ElementOutput, cutlass::layout::RowMajor,
      ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm80,
      cutlass::gemm::GemmShape<64, 64, 512>,
      cutlass::gemm::GemmShape<32, 32, 512>, cutlass::gemm::GemmShape<16, 8, 256>,
      cutlass::epilogue::thread::LinearCombination<
          ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
          ElementAccumulator, ElementCompute>,
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 6, 128, 128,
      false, cutlass::arch::OpXorPopc>;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
} )

////////////////////////////////////////////////////////////////////////////////

#endif  // #if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
