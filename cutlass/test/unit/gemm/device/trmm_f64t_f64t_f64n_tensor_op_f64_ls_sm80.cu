/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide TRMM interface

  
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/blas3.h"
#include "cutlass/gemm/device/trmm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/trmm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed_trmm_universal.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_l_nu_tensor_op_f64, 32x32x16_16x16x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kLower,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 16, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_l_nu_tensor_op_f64, 64x64x16_32x32x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kLower,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_l_nu_tensor_op_f64, 128x64x16_64x32x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kLower,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 16>,
    cutlass::gemm::GemmShape<64, 32, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_l_nu_tensor_op_f64, 64x128x16_32x64x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kLower,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 16>,
    cutlass::gemm::GemmShape<32, 64, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;
  
  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_l_nu_tensor_op_f64, 128x128x16_32x64x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kLower,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 16>,
    cutlass::gemm::GemmShape<32, 64, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_u_nu_tensor_op_f64, 32x32x16_16x16x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kUpper,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 16, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_u_nu_tensor_op_f64, 64x64x16_32x32x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kUpper,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_u_nu_tensor_op_f64, 128x64x16_64x32x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kUpper,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 16>,
    cutlass::gemm::GemmShape<64, 32, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_u_nu_tensor_op_f64, 64x128x16_32x64x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kUpper,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 16>,
    cutlass::gemm::GemmShape<32, 64, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;
  
  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Trmm_f64t_f64t_f64n_ls_u_nu_tensor_op_f64, 128x128x16_32x64x16) {

  using ElementOutput = double;
  using ElementAccumulator = double;

  using Trmm = cutlass::gemm::device::Trmm<
    double,
    cutlass::layout::RowMajor,
    cutlass::SideMode::kLeft,
    cutlass::FillMode::kUpper,
    cutlass::DiagType::kNonUnit,
    double,
    cutlass::layout::RowMajor,
    ElementOutput,
    cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 16>,
    cutlass::gemm::GemmShape<32, 64, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      1,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllTrmmUniversal<Trmm>());
}
/////////////////////////////////////////////////////////////////////////////////////////////////

#endif // #if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
