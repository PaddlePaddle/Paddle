#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_complex.h"


#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed_complex.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
//  Operands data type: complex<float>
//  Rounding: float -> tfloat32_t (round to nearest)
//  Instruction operand data type: tfloat32_t (real part) and  tfloat32_t (imaginary part)
//  Math instruction: MMA.1688.F32.TF32
//  Instruction output/accumulation data type: f32 (real part) and f32 (imaginary part)
//  Output data type: complex<float>
/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_cf32t_cf32n_cf32t_tensor_op_tf32_f32, 32x32x16_16x16x16) {

  using Element = cutlass::complex<float>;

  using Gemm = cutlass::gemm::device::GemmComplex<
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::layout::ColumnMajor,
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 16, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      Element,
      1,
      Element,
      Element
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemmComplex<Gemm>());
}


/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_cf32t_cf32n_cf32t_tensor_op_tf32_f32, 64x64x16_16x32x16) {
  
  using Element = cutlass::complex<float>;

  using Gemm = cutlass::gemm::device::GemmComplex<
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::layout::ColumnMajor,
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<16, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      Element,
      1,
      Element,
      Element
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemmComplex<Gemm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_cf32t_cf32n_cf32t_tensor_op_tf32_f32, 64x64x16_32x32x16) {

  
  using Element = cutlass::complex<float>;

  using Gemm = cutlass::gemm::device::GemmComplex<
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::layout::ColumnMajor,
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      Element,
      1,
      Element,
      Element
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemmComplex<Gemm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_cf32t_cf32n_cf32t_tensor_op_tf32_f32, 128x64x16_64x32x16) {

  using Element = cutlass::complex<float>;;

  using Gemm = cutlass::gemm::device::GemmComplex<
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::layout::ColumnMajor,
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 16>,
    cutlass::gemm::GemmShape<64, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      Element,
      1,
      Element,
      Element
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    4
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemmComplex<Gemm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_cf32t_cf32n_cf32t_tensor_op_tf32_f32, 64x128x16_32x64x16) {

  using Element = cutlass::complex<float>;;

  using Gemm = cutlass::gemm::device::GemmComplex<
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::layout::ColumnMajor,
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 16>,
    cutlass::gemm::GemmShape<32, 64, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      Element,
      1,
      Element,
      Element
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;
  
  EXPECT_TRUE(test::gemm::device::TestAllGemmComplex<Gemm>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_cf32t_cf32n_cf32t_tensor_op_tf32_f32, 128x128x16_32x64x16) {

  using Element = cutlass::complex<float>;;

  using Gemm = cutlass::gemm::device::GemmComplex<
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::layout::ColumnMajor,
    Element,
    cutlass::layout::RowMajor,
    Element,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 16>,
    cutlass::gemm::GemmShape<32, 64, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
      Element,
      1,
      Element,
      Element
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemmComplex<Gemm>());
}

#endif // #if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
