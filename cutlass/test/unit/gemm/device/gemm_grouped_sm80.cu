#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface
    
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/cutlass.h"

#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed_grouped.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Visitor class to abstract away the algorithm for iterating over tiles.
//
// This is the prototype. We will delete this when the efficient kernel is
// available.
struct GemmGroupedProblemVisitor {

  struct Params {
    cutlass::gemm::GemmCoord const *problem_sizes;
    int32_t                         problem_count;
    int64_t const                  *tile_count;
  };

  struct SharedStorage {
    //
    // Nothing for now. As an optimization step, we could consider parallel
    // argmin or prefix sums across the block.
    //
  };

  //
  // Data members
  //
  
  SharedStorage &shared_storage;
  Params const &params;
  cutlass::MatrixCoord threadblock_shape;

  int64_t tile_idx;
  int64_t tile_count_sum;
  int64_t problem_tile_start;
  int32_t problem_idx;

  //
  // Methods
  //
  CUTLASS_DEVICE
  GemmGroupedProblemVisitor(
    SharedStorage &shared_storage_, 
    Params const &params_,
    cutlass::MatrixCoord threadblock_shape_,
    int32_t block_idx
  ):
    shared_storage(shared_storage_),
    params(params_),
    threadblock_shape(threadblock_shape_),
    tile_idx(block_idx),
    tile_count_sum(0),
    problem_idx(0)
  {

    cutlass::gemm::GemmCoord problem = params.problem_sizes[problem_idx];

    cutlass::gemm::GemmCoord  grid = grid_shape(problem);

    problem_tile_start = 0;
    tile_count_sum = grid.m() * grid.n();
  }

  /// Get the grid shape
  CUTLASS_HOST_DEVICE
  static cutlass::gemm::GemmCoord grid_shape(
    cutlass::gemm::GemmCoord const &problem,
    cutlass::MatrixCoord const & block_shape) {

    return cutlass::gemm::GemmCoord(
      ((problem.m() - 1 + block_shape.row()) / block_shape.row()),
      ((problem.n() - 1 + block_shape.column()) / block_shape.column()),
      1);
  }

  /// Get the grid shape
  CUTLASS_DEVICE
  cutlass::gemm::GemmCoord grid_shape(cutlass::gemm::GemmCoord const &problem) const {
    return grid_shape(problem, threadblock_shape);
  }

  /// Returns true if there is a tile to compute
  CUTLASS_DEVICE
  bool next_tile() {

    if (tile_idx < tile_count_sum) {
      return true;
    }

    do {
      ++problem_idx;

      if (problem_idx >= params.problem_count) {
        return false;
      }

      cutlass::gemm::GemmCoord problem = params.problem_sizes[problem_idx];
      cutlass::gemm::GemmCoord  grid = grid_shape(problem);

      int64_t tile_count = grid.m() * grid.n();

      problem_tile_start = tile_count_sum;
      tile_count_sum += tile_count;

    } while (tile_count_sum <= tile_idx);

    return true;
  }

  /// Gets the global tile index
  CUTLASS_HOST_DEVICE
  int64_t tile_index() const {
    return tile_idx;
  }

  /// Gets the index of the problem
  CUTLASS_HOST_DEVICE
  int32_t problem_index() const {
    return problem_idx;
  }

  /// Returns the problem size for the current problem
  CUTLASS_HOST_DEVICE
  cutlass::gemm::GemmCoord problem_size() const {
    return params.problem_sizes[problem_idx];
  }

  CUTLASS_HOST_DEVICE
  int64_t threadblock_index() const {
    return tile_idx - problem_tile_start;
  }

  CUTLASS_DEVICE
  void advance(int32_t grid_size) {
    tile_idx += grid_size; 
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template <int CtaShapeM, int CtaShapeN>
__global__ void GroupedBatchedKernel(GemmGroupedProblemVisitor::Params params) {

  __shared__ GemmGroupedProblemVisitor::SharedStorage shared_storage;

  GemmGroupedProblemVisitor problem_visitor(
    shared_storage, 
    params, 
    {CtaShapeM, CtaShapeN}, 
    blockIdx.x);

  while (problem_visitor.next_tile()) {

    cutlass::gemm::GemmCoord problem_size = problem_visitor.problem_size();
    int64_t cta_idx                       = problem_visitor.threadblock_index();

    cutlass::gemm::GemmCoord grid_shape = problem_visitor.grid_shape(problem_size);

    int cta_tile_m_idx = int(cta_idx / grid_shape.n());
    int cta_tile_n_idx = int(cta_idx % grid_shape.n());

    //
    // Do the MMA
    //

    if (threadIdx.x == 0) {
      #if 0
      printf("Block %d - tile: %lld, problem %d, cta_idx: %lld, cta(m: %d, n: %d)\n", 
        blockIdx.x, 
        problem_visitor.tile_index(), 
        problem_visitor.problem_index(), 
        cta_idx, 
        cta_tile_m_idx, 
        cta_tile_n_idx);
      #endif
    }

    // Next tile
    problem_visitor.advance(gridDim.x);
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_scheduler, 64x64x32_32x32x32) {

  int32_t problem_count = 16;

  int const kCtaShapeM = 64;
  int const kCtaShapeN = 64;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes(problem_count);
  std::vector<int64_t> tile_counts(problem_count);

  // construct a few problems of random sizes
  srand(1921);
  for (int32_t i = 0; i < problem_count; ++i) {
    problem_sizes.at(i) = cutlass::gemm::GemmCoord(
      8 * (rand() % 48) + 64,
      8 * (rand() % 48) + 64,
      8 * (rand() % 48) + 64);
  }

  // compute prefix sum
  int64_t tile_count = 0;

  for (int32_t i = 0; i < problem_count; ++i) {

    cutlass::gemm::GemmCoord grid_shape = GemmGroupedProblemVisitor::grid_shape(
      problem_sizes.at(i), {kCtaShapeM, kCtaShapeN});

    int32_t problem_tile_count = (grid_shape.m() * grid_shape.n());

    int64_t tile_start = tile_count;

    tile_count += problem_tile_count;
    tile_counts.at(i) = tile_count;

    if (false) {
      std::cout << "Problem " << i << " size(" 
        << problem_sizes.at(i).m() << "-by-" << problem_sizes.at(i).n() 
        << ") - tiles: " << problem_tile_count << ",  grid(" << grid_shape.m() << ", " << grid_shape.n() 
        << "), tiles[" << tile_start << ", " << tile_count << ")" << std::endl;  
    }
  }

  // Copy to device memory
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device(problem_count);
  cutlass::DeviceAllocation<int64_t>                  tile_counts_device(problem_count);

  problem_sizes_device.copy_from_host(problem_sizes.data());
  tile_counts_device.copy_from_host(tile_counts.data());

  GemmGroupedProblemVisitor::Params params;
  params.problem_sizes = problem_sizes_device.get();
  params.problem_count = problem_count;
  params.tile_count = tile_counts_device.get();

  // Launch the kernel
  dim3 grid(108, 1, 1);
  dim3 block(128, 1, 1);

  GroupedBatchedKernel<kCtaShapeM, kCtaShapeN><<< grid, block >>>(params);

  // wait
  hipDeviceSynchronize();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f16n_f16t_f32n_tensor_op_f32, 128x128x32_64x64x32) {

  using ElementOutput = float;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    cutlass::half_t, 
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kNone,
    8,
    cutlass::half_t,
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kNone,
    8,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>, 
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    3>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(24);
  EXPECT_TRUE(passed);
  
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f16n_f16t_f32t_tensor_op_f32, 128x128x32_64x64x32) {

  using ElementOutput = float;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    cutlass::ComplexTransform::kNone,
    8,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    cutlass::ComplexTransform::kNone,
    8,
    ElementOutput, cutlass::layout::RowMajor,    // row major
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
    3>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(24);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f16t_f16n_f32n_tensor_op_f32, 128x64x32_64x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    cutlass::half_t, 
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kNone,
    8,
    cutlass::half_t,
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kNone,
    8,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32>, 
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    4>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f16t_f16n_f32t_tensor_op_f32, 128x64x32_64x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    8,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    cutlass::ComplexTransform::kNone,
    8,
    ElementOutput, cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 128 / cutlass::sizeof_bits<ElementOutput>::value,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
    4>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f64t_f64t_f64n_tensor_op_f64, 64x64x16_32x32x16) {

  using ElementInput = double;
  using ElementOutput = double;
  using ElementAccumulator = double;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput, 
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    4>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f32t_f32t_f32n_simt_f32, 128x128x8_64x32x1) {

  using ElementInput = float;
  using ElementOutput = float;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput, 
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassSimt, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 8>,
    cutlass::gemm::GemmShape<64, 32, 8>,
    cutlass::gemm::GemmShape<1, 1, 1>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    3>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f32t_f32t_f32t_simt_f32, 128x128x8_64x32x1) {

  using ElementInput = float;
  using ElementOutput = float;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    1,
    ElementOutput, cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassSimt,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 8>,
    cutlass::gemm::GemmShape<64, 32, 8>,
    cutlass::gemm::GemmShape<1, 1, 1>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
    3>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f32t_f32t_f32n_simt_f32, 128x64x8_64x32x1) {

  using ElementInput = float;
  using ElementOutput = float;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    1,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator,
    cutlass::arch::OpClassSimt,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 8>,
    cutlass::gemm::GemmShape<64, 32, 8>,
    cutlass::gemm::GemmShape<1, 1, 1>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
    3>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_f32t_f32t_f32t_simt_f32, 128x64x8_64x32x1) {

  using ElementInput = float;
  using ElementOutput = float;
  using ElementAccumulator = float;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::RowMajor,
    cutlass::ComplexTransform::kNone,
    1,
    ElementOutput, cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassSimt,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 8>,
    cutlass::gemm::GemmShape<64, 32, 8>,
    cutlass::gemm::GemmShape<1, 1, 1>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
    3>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_cf32n_cf32n_cf32n_tensorop_f32, 64x64x16_32x32x16) {

  using ElementInput = cutlass::complex<float>;
  using ElementOutput = cutlass::complex<float>;
  using ElementAccumulator = cutlass::complex<float>;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput, 
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    3,
    cutlass::arch::OpMultiplyAddComplex>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_cf32c_cf32t_cf32n_tensorop_f32, 64x64x16_32x32x16) {

  using ElementInput = cutlass::complex<float>;
  using ElementOutput = cutlass::complex<float>;
  using ElementAccumulator = cutlass::complex<float>;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput, 
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kConjugate,
    1,
    ElementInput,
    cutlass::layout::ColumnMajor, 
    cutlass::ComplexTransform::kConjugate,
    1,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    3,
    cutlass::arch::OpMultiplyAddComplex>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_cf32c_cf32t_cf32t_tensorop_f32, 64x64x16_32x32x16) {

  using ElementInput = cutlass::complex<float>;
  using ElementOutput = cutlass::complex<float>;
  using ElementAccumulator = cutlass::complex<float>;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput,
    cutlass::layout::ColumnMajor,
    cutlass::ComplexTransform::kConjugate,
    1,
    ElementInput,
    cutlass::layout::ColumnMajor,
    cutlass::ComplexTransform::kConjugate,
    1,
    ElementOutput, cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 16>,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 8, 8>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
    3,
    cutlass::arch::OpMultiplyAddComplex>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_GemmGrouped_cf32t_cf32h_cf32n_tensorop_f32, 64x64x16_16x16x16) {

  using ElementInput = cutlass::complex<double>;
  using ElementOutput = cutlass::complex<double>;
  using ElementAccumulator = cutlass::complex<double>;

  using GemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
    ElementInput, 
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kNone,
    1,
    ElementInput,
    cutlass::layout::RowMajor, 
    cutlass::ComplexTransform::kConjugate,
    1,
    ElementOutput, cutlass::layout::ColumnMajor,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 32, 16>,
    cutlass::gemm::GemmShape<16, 16, 16>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    cutlass::epilogue::thread::LinearCombination<
        ElementOutput, 1,
        ElementAccumulator, ElementAccumulator>,
    cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle, 
    3,
    cutlass::arch::OpMultiplyAddComplex>::GemmKernel;

  using Gemm = cutlass::gemm::device::GemmGrouped<GemmKernel>;

  //
  // Test
  //

  test::gemm::device::TestbedGrouped<Gemm> testbed;

  bool passed = testbed.run(27);
  EXPECT_TRUE(passed);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#endif // #if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
