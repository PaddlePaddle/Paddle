#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for thread-level GEMM
*/

#include "../../common/cutlass_unit_test.h"

#include "cutlass/epilogue/epilogue_workspace.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace gemm {
namespace threadblock {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Kernel computes accumulator data and stores it out
template <typename Epilogue>
__global__ void kernel_epilogue_workspace(typename Epilogue::Params params) {

  __shared__ typename Epilogue::SharedStorage shared_storage;

  int warp_id = threadIdx.y;
  int lane_id = threadIdx.x;

  Epilogue epilogue(params, shared_storage, warp_id, lane_id);

  //
  // Initialize accumulator tile
  //
  typename Epilogue::FragmentC accum;

  CUTLASS_PRAGMA_UNROLL
  for (int i = 0; i < Epilogue::FragmentC::kElements; ++i) {
    accum[i] = Element(warp_id * blockDim.x + lane_id);
  }

  //
  // Efficient epilogue
  //

  cutlass::GemmCoord tb_tile_coord{blockIdx.x, blockIdx.y, 0};
  
  cutlass::GemmCoord problem_size = 
    tb_tile_coord * 
    cutlass::GemmCoord{Epilogue::Shape::kM, Epilogue::Shape::kN, 1};

  // Store accumulators
  epilogue(
    problem_size, 
    tb_tile_coord, 
    accum);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace threadblock
} // namespace gemm
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM75_gemm_threadblock_epilogue_workspace, tensor_op_128x128_64x64) {

  //
  // Define an instance of the epilogue and see if it works
  //
  static int const kWarpCount = 4;
  static int const kWarpSize = 32;

  using Shape = cutlass::MatrixShape<128, 128>;
  using FragmentC = cutlass::Array<int, Shape::kCount / (kWarpCount * kWarpSize)>;

  using Epilogue = cutlass::gemm::threadblock::EpilogueWorkspace<
    Shape,
    kWarpCount,
    FragmentC
  >;

  typename Epilogue::Params params(
    
  );

  // Launch the kernel
  dim3 grid(1,1);
  dim3 block(kWarpSize, kWarpCount);

  test::gemm::threadblock::kernel_epilogue_workspace<Epilogue><<< grid, block >>>(
    params
  );

  hipError_t result = hipDeviceSynchronize();
  EXPECT_EQ(result, hipSuccess) << "Kernel launch error - " << hipGetErrorString(result);

  //
  // 
  //
}

/////////////////////////////////////////////////////////////////////////////////////////////////
