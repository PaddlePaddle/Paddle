#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for TensorReduce family of device-wide operators
*/

#include <iostream>
#include <limits>

#include "../../common/cutlass_unit_test.h"

#include "cutlass/cutlass.h"
#include "cutlass/complex.h"
#include "cutlass/reduction/thread/reduction_operators.h"
#include "cutlass/reduction/device/tensor_reduce.h"

#include "cutlass/functional.h"
#include "cutlass/layout/tensor.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/tensor_view_io.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// This reduces the W dimension, transforming an NHWC tensor into NHWC with W=1.
template <
  typename TensorReduction, 
  typename ElementCompute = typename TensorReduction::ElementCompute
>
bool TestAllReduction_NHWC_reduce_w(ElementCompute reduction_identity = ElementCompute()) {

  using Layout = typename TensorReduction::Layout;
  using ElementOutput = typename TensorReduction::ElementOutput;
  using ElementSource = typename TensorReduction::ElementSource;

  int const kV = TensorReduction::kVectorLength;

  int const N_indices[] = {1, 2, 5, 10};
  int const H_indices[] = {1, 3, 9 };
  int const W_indices[] = {1, 5, 19, 40, 224};
  int const C_indices[] = {
    kV, 
    2 * kV, 
    5 * kV, 
    9 * kV, 
    17 * kV, 
    39 * kV, 
    257 * kV, 
    kV * 760
  };

  using Element = int;

  for (int N : N_indices) {
    for (int H : H_indices) {
      for (int W : W_indices) {
        for (int C : C_indices) {

          cutlass::HostTensor<ElementSource, Layout> src_tensor({N, H, W, C});
          cutlass::HostTensor<ElementOutput, Layout> dst_tensor({N, H, 1, C});

          cutlass::reference::host::TensorFillRandomUniform(
            src_tensor.host_view(), 17, 10, -10, 0);

          cutlass::reference::host::BlockFillSequential(
            dst_tensor.host_data(), dst_tensor.capacity());

          dst_tensor.sync_device();
          src_tensor.sync_device();

          // Execute a tensor reduction over rank 2 (the 'W' dimension is reduced; NHWC => NHC)
          TensorReduction reduction(src_tensor.extent(), 2);

          cutlass::DeviceAllocation<uint8_t> device_workspace(reduction.workspace_size());

          cutlass::Status status = reduction.reduce(
            dst_tensor.device_ref(),
            src_tensor.device_ref(),
            device_workspace.get(),
            reduction_identity
          );

          EXPECT_EQ(status, cutlass::Status::kSuccess);
          EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
          // Reference check
          dst_tensor.sync_host();

          typename TensorReduction::ReductionOp reduction_op;

          for (int n = 0; n < src_tensor.extent().n(); ++n) {
            for (int h = 0; h < src_tensor.extent().h(); ++h) {
              for (int c = 0; c < src_tensor.extent().c(); ++c) {

                ElementCompute w_accum = reduction_identity;

                for (int w = 0; w < src_tensor.extent().w(); ++w) {
                  w_accum = reduction_op(w_accum, ElementCompute(src_tensor.at({n, h, w, c})));
                }

                ElementCompute got = ElementCompute(dst_tensor.at({n, h, 0, c}));

                bool equal = (w_accum == got);

                EXPECT_TRUE(equal);
                if (!equal) {

                  std::cerr 
                    << "Error at location (" << n << ", " << h << ", 0, " << c << ")" << std::endl;

                  std::cerr 
                    << "  expected: " << w_accum << std::endl
                    << "       got: " << got << std::endl;

                  std::cerr 
                    << "Problem: " << src_tensor.extent() << " -> " 
                    << dst_tensor.extent() << std::endl;

                  std::cerr 
                    << "   Grid: " << reduction.reduction_strided.grid_shape 
                    << "\n  Block: " << reduction.reduction_strided.threadblock_shape << std::endl
                    << "  Final: " << reduction.reduction_strided.grid_final 
                    << "\n  Block: " << reduction.reduction_strided.threadblock_final << "\n";

                  return false;
                }
              }
            }
          }
        }
      }
    }
  }

  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_reduce_w_f32x8_f16x8) {

  int const kV = 8;
  using ElementOutput = float;
  using ElementSource = cutlass::half_t;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_reduce_w_f32x2_f16x2) {

  int const kV = 2;
  using ElementOutput = float;
  using ElementSource = cutlass::half_t;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_reduce_w_f32x1_f16x1) {

  int const kV = 1;
  using ElementOutput = float;
  using ElementSource = cutlass::half_t;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_reduce_w_s32x4) {

  int const kV = 4;
  using Element = int;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::plus<Element>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    Element,
    Element,
    Layout,
    Functor,
    kV,
    Element
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_reduce_w_cf32) {

  int const kV = 1;
  using ElementOutput = cutlass::complex<float>;
  using ElementSource = cutlass::complex<float>;
  using ElementCompute = cutlass::complex<float>;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_maximum_w_cf32) {

  int const kV = 1;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::maximum<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>( -std::numeric_limits<float>::max() ));          
}

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_minimum_w_cf32) {

  int const kV = 1;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::minimum<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>(std::numeric_limits<float>::max()));          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_XOR_w_u32) {

  int const kV = 1;
  using ElementOutput = int;
  using ElementSource = int;
  using ElementCompute = int;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::bit_xor<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_AND_w_s32) {

  int const kV = 1;
  using ElementOutput = unsigned;
  using ElementSource = unsigned;
  using ElementCompute = unsigned;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::bit_and<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>(0xffffffff));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_OR_w_u32) {

  int const kV = 1;
  using ElementOutput = int;
  using ElementSource = int;
  using ElementCompute = int;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::bit_or<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>());          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_ANY_w_s32) {

  int const kV = 1;
  using ElementOutput = int;
  using ElementSource = int;
  using ElementCompute = int;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::logical_or<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>(ElementCompute(0)));          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_ALL_w_s32) {

  int const kV = 1;
  using ElementOutput = int;
  using ElementSource = int;
  using ElementCompute = int;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::logical_and<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>(ElementCompute(1)));          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_ANY_w_f32) {

  int const kV = 1;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::logical_or<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>(ElementCompute(0)));          
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHC
TEST(Reduction_TensorReduce, nhwc_ALL_w_f32) {

  int const kV = 1;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  using Layout = cutlass::layout::TensorNHWC;

  // Define the functor
  using Functor = cutlass::logical_and<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;

  EXPECT_TRUE(TestAllReduction_NHWC_reduce_w<TensorReduction>(ElementCompute(1)));          
}

/////////////////////////////////////////////////////////////////////////////////////////////////
