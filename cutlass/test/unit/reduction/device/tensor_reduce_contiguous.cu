#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for TensorReduce family of device-wide operators
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"

#include "cutlass/cutlass.h"
#include "cutlass/complex.h"
#include "cutlass/reduction/thread/reduction_operators.h"
#include "cutlass/reduction/device/tensor_reduce.h"

#include "cutlass/functional.h"
#include "cutlass/layout/tensor.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/tensor_view_io.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// This reduces the C dimension, transforming an NHWC tensor into NHWC with C=1.
template <typename TensorReduction, typename ElementCompute = typename TensorReduction::ElementCompute>
bool TestAllReduction_NHWC_reduce_c(ElementCompute reduction_identity = ElementCompute()) {

  using Layout = typename TensorReduction::Layout;
  using ElementOutput = typename TensorReduction::ElementOutput;
  using ElementSource = typename TensorReduction::ElementSource;

  int const kV = TensorReduction::kVectorLength;

  int const N_indices[] = {3, 13};
  int const H_indices[] = {5, 17};
  int const W_indices[] = {7, 19};
  int const C_indices[] = {2049, 2048, 2047, 384, 64, 48, 32, 24, 16, 12, 8, 6, 4, 3, 2, 1};
  
  for (int N : N_indices) {
    for (int H : H_indices) {
      for (int W : W_indices) {
        for (int Cx : C_indices) {

          int C = Cx * kV;

          cutlass::HostTensor<ElementSource, Layout> src_tensor({N, H, W, C});
          cutlass::HostTensor<ElementOutput, Layout> dst_tensor({N, H, W, 1});

          cutlass::reference::host::TensorFillRandomUniform(
            src_tensor.host_view(), 17, 10, -10, 0);

          dst_tensor.sync_device();
          src_tensor.sync_device();

          // Execute a tensor reduction over rank 3 (the 'C' dimension is reduced; NHWC => NHW)
          TensorReduction reduction(src_tensor.extent(), 3);

          cutlass::DeviceAllocation<uint8_t> device_workspace(reduction.workspace_size());

          cutlass::Status status = reduction.reduce(
            dst_tensor.device_ref(),
            src_tensor.device_ref(),
            device_workspace.get(),
            reduction_identity
          );

          EXPECT_EQ(status, cutlass::Status::kSuccess);
          EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
          
          dst_tensor.sync_host();

          typename TensorReduction::ReductionOp reduction_op;

          //
          // Reference check
          //
          for (int n = 0; n < src_tensor.extent().n(); ++n) {
            for (int h = 0; h < src_tensor.extent().h(); ++h) {
              for (int w = 0; w < src_tensor.extent().w(); ++w) {

                ElementCompute c_accum = reduction_identity;

                for (int c = 0; c < src_tensor.extent().c(); ++c) {
                  c_accum = reduction_op(c_accum, ElementCompute(src_tensor.at({n, h, w, c})));
                }

                ElementCompute got = ElementCompute(dst_tensor.at({n, h, w, 0}));

                bool equal = (c_accum == got);

                EXPECT_TRUE(equal);
                if (!equal) {

                  std::cerr 
                    << "Error at location (" << n << ", " << h << ", " << w << ", 0)" << std::endl;

                  std::cerr 
                    << "  expected: " << c_accum << std::endl
                    << "       got: " << got << std::endl;

                  std::cerr 
                    << "Problem: " << src_tensor.extent() << " -> " 
                    << dst_tensor.extent() << std::endl;

                  std::cerr 
                    << "   Grid: " << reduction.reduction_strided.grid_shape 
                    << "\n   Block: " << reduction.reduction_strided.threadblock_shape << std::endl
                    << "  FInal: " << reduction.reduction_strided.grid_final 
                    << "\n   Block: " << reduction.reduction_strided.threadblock_final << "\n";

                  return false;
                }

              } //w
            } // h
          } // n
          
          //
          // Next problem
          //

        } // C
      } // W
    } // H
  } // N

  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_reduce_c_f32x1) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 1;
  
  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_reduce_c_f32x1_f16x1) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = cutlass::half_t;
  using ElementCompute = float;
  int const kV = 1;
  
  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_reduce_c_f32x2) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 2;
  
  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_reduce_c_f32x2_f16x2) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = cutlass::half_t;
  using ElementCompute = float;
  int const kV = 2;
  
  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_reduce_c_f32x4) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 4;
  
  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_reduce_c_f32x4_f16x4) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = cutlass::half_t;
  using ElementCompute = float;
  int const kV = 4;
  
  // Define the functor
  using Functor = cutlass::plus<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_maximum_c_f32x4) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 4;
  
  // Define the functor
  using Functor = cutlass::maximum<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>( -std::numeric_limits<float>::max() ));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_minimum_c_f32x4) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 4;
  
  // Define the functor
  using Functor = cutlass::minimum<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>( std::numeric_limits<float>::max() ));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_ANY_c_s32) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = int;
  using ElementSource = int;
  using ElementCompute = int;
  int const kV = 1;
  
  // Define the functor
  using Functor = cutlass::logical_or<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>( ElementCompute(0) ));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_ALL_c_s32) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = int;
  using ElementSource = int;
  using ElementCompute = int;
  int const kV = 1;
  
  // Define the functor
  using Functor = cutlass::logical_and<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>( ElementCompute(1) ));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_ANY_c_f32) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 1;
  
  // Define the functor
  using Functor = cutlass::logical_or<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>( ElementCompute(0) ));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Test tensor reduction from NHWC to NHW
TEST(Reduction_TensorReduce, nhwc_ALL_c_f32) {

  using Layout = cutlass::layout::TensorNHWC;
  using ElementOutput = float;
  using ElementSource = float;
  using ElementCompute = float;
  int const kV = 1;
  
  // Define the functor
  using Functor = cutlass::logical_and<ElementCompute>;

  using TensorReduction = cutlass::reduction::device::TensorReduction<
    ElementOutput,
    ElementSource,
    Layout,
    Functor,
    kV,
    ElementCompute
  >;
  
  EXPECT_TRUE(TestAllReduction_NHWC_reduce_c<TensorReduction>( ElementCompute(1) ));
}

/////////////////////////////////////////////////////////////////////////////////////////////////
