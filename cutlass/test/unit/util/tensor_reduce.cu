#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <complex>

#include "../common/cutlass_unit_test.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/layout/tensor.h"

#include "cutlass/util/reference/device/tensor_reduce.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/host_tensor.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(TensorReduce, norm_rowmajor_f32) {

  int const kM = 129;
  int const kN = 91;

  cutlass::HostTensor<float, cutlass::layout::RowMajor> tensor({kM, kN});

  for (int m = 0; m < kM; ++m) {
    for (int n = 0; n < kN; ++n) {

      float x = float(((m * kN + m + 7) % 8) - 4);

      tensor.at({m, n}) = x;
    }
  }

  tensor.sync_device();

  double device_norm = cutlass::reference::device::TensorNorm(tensor.device_view(), double());
  double host_norm = cutlass::reference::host::TensorNorm(tensor.host_view(), double());

  EXPECT_TRUE(std::abs(host_norm - device_norm) < 0.001);
}

TEST(TensorReduce, norm_nhwc_f32) {

  int const kN = 19;
  int const kH = 18;
  int const kW = 17;
  int const kC = 16;

  cutlass::HostTensor<float, cutlass::layout::TensorNHWC> tensor({kN, kH, kW, kC});

  int idx = 0;

  double computed_norm = double();

  for (int n = 0; n < kN; ++n) {
    for (int h = 0; h < kH; ++h) {
      for (int w = 0; w < kW; ++w) {
        for (int c = 0; c < kC; ++c, ++idx) {
      
          float x = float(((idx + 7) % 8) - 4);

          computed_norm += double(x) * double(x);

          tensor.at({n, h, w, c}) = x;
        }
      }
    }
  }

  computed_norm = std::sqrt(computed_norm);

  tensor.sync_device();

  double device_norm = cutlass::reference::device::TensorNorm(tensor.device_view(), double());
  double host_norm = cutlass::reference::host::TensorNorm(tensor.host_view(), double());

  EXPECT_TRUE(std::abs(host_norm - device_norm) < 0.001 && std::abs(computed_norm - host_norm) < 0.001)
    << "computed norm: " << computed_norm << "\n"
    << " host norm: " << host_norm << "\n"
    << "device norm: " << device_norm << "\n";
}

TEST(TensorReduce, norm_nhwc_f16) {

  int const kN = 69;
  int const kH = 68;
  int const kW = 67;
  int const kC = 66;

  cutlass::HostTensor<cutlass::half_t, cutlass::layout::TensorNHWC> tensor({kN, kH, kW, kC});

  int idx = 0;

  double computed_norm = double();

  for (int n = 0; n < kN; ++n) {
    for (int h = 0; h < kH; ++h) {
      for (int w = 0; w < kW; ++w) {
        for (int c = 0; c < kC; ++c, ++idx) {
      
          float x = float(((idx + 7) % 8) - 4);
          computed_norm += double(x) * double(x);

          tensor.at({n, h, w, c}) = cutlass::half_t(x);
        }
      }
    }
  }
  
  computed_norm = std::sqrt(computed_norm);

  tensor.sync_device();

  double device_norm = cutlass::reference::device::TensorNorm(tensor.device_view(), double());
  double host_norm = cutlass::reference::host::TensorNorm(tensor.host_view(), double());

  EXPECT_TRUE(std::abs(host_norm - device_norm) < 0.001 && std::abs(computed_norm - host_norm) < 0.001)
    << "computed norm: " << computed_norm << "\n"
    << " host norm: " << host_norm << "\n"
    << "device norm: " << device_norm << "\n";
}

TEST(TensorReduce, norm_diff_nhwc_f32) {

  int const kN = 59;
  int const kH = 24;
  int const kW = 57;
  int const kC = 78;

  using Layout = cutlass::layout::TensorNHWC;

  cutlass::HostTensor<float, Layout> tensor_A({kN, kH, kW, kC});
  cutlass::HostTensor<float, Layout> tensor_B({kN, kH, kW, kC});


  int idx = 0;

  double sum_sq_diff = 0;

  for (int n = 0; n < kN; ++n) {
    for (int h = 0; h < kH; ++h) {
      for (int w = 0; w < kW; ++w) {
        for (int c = 0; c < kC; ++c, ++idx) {
      
          float a = float(((idx * 5 + 7) % 8) - 4);
          float b = float(((idx * 3 + 7) % 8) - 4);

          sum_sq_diff += double(a - b) * double(a - b);

          tensor_A.at({n, h, w, c}) = a;
          tensor_B.at({n, h, w, c}) = b;
        }
      }
    }
  }

  tensor_A.sync_device();
  tensor_B.sync_device();

  double device_norm = cutlass::reference::device::TensorNormDiff(
    tensor_A.device_view(), tensor_B.device_view(), double());

  double host_norm = std::sqrt(sum_sq_diff);
  
  EXPECT_TRUE(std::abs(host_norm - device_norm) < 0.001f)
    << "  host norm: " << host_norm << "\n"
    << "device norm: " << device_norm;
}


TEST(TensorReduce, norm_diff_nhwc_f16) {

  int const kN = 59;
  int const kH = 24;
  int const kW = 57;
  int const kC = 78;

  using Layout = cutlass::layout::TensorNHWC;

  cutlass::HostTensor<cutlass::half_t, Layout> tensor_A({kN, kH, kW, kC});
  cutlass::HostTensor<cutlass::half_t, Layout> tensor_B({kN, kH, kW, kC});

  int idx = 0;

  double sum_sq_diff = 0;

  for (int n = 0; n < kN; ++n) {
    for (int h = 0; h < kH; ++h) {
      for (int w = 0; w < kW; ++w) {
        for (int c = 0; c < kC; ++c, ++idx) {
      
          float a = float(((idx * 5 + 7) % 8) - 4);
          float b = float(((idx * 3 + 7) % 8) - 4);

          sum_sq_diff += double(a - b) * double(a - b);

          tensor_A.at({n, h, w, c}) = cutlass::half_t(a);
          tensor_B.at({n, h, w, c}) = cutlass::half_t(b);
        }
      }
    }
  }

  tensor_A.sync_device();
  tensor_B.sync_device();

  double device_norm = cutlass::reference::device::TensorNormDiff(
    tensor_A.device_view(), tensor_B.device_view(), double());

  double host_norm = std::sqrt(sum_sq_diff);
  
  EXPECT_TRUE(std::abs(host_norm - device_norm) < 0.001f)
    << "  host norm: " << host_norm << "\n"
    << "device norm: " << device_norm;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

