#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests cutlass::transform::threadblock::PredicatedTileIterator 
*/

#include "../../common/cutlass_unit_test.h"

#include "cutlass/cutlass.h"

#include "cutlass/transform/pitch_linear_thread_map.h"
#include "cutlass/transform/threadblock/predicated_tile_iterator.h"
#include "cutlass/transform/threadblock/predicated_tile_iterator_2dthreadtile.h"

#include "cutlass/util/tensor_view_io.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace transform {
namespace threadblock {
namespace kernel {

/// Copy with an iterator
template <typename Iterator>
__global__ void copy(
  typename Iterator::Params dst_params, 
  typename Iterator::Element *dst_pointer,
  typename Iterator::Params src_params,
  typename Iterator::Element *src_pointer,
  cutlass::Coord<2> extent) {

  Iterator dst_iterator(dst_params, dst_pointer, extent, threadIdx.x);
  Iterator src_iterator(src_params, src_pointer, extent, threadIdx.x);

  int iterations = (extent[1] + Iterator::Shape::kStrided - 1) / Iterator::Shape::kStrided;

  typename Iterator::Fragment frag;

  for(int i = 0; i < frag.size(); i++)
    frag[i] = 0;

  src_iterator.load(frag);
  dst_iterator.store(frag);

  ++dst_iterator;
  ++src_iterator;

  for (; iterations > 1; --iterations) {
    
    src_iterator.load(frag);
    dst_iterator.store(frag);

    ++dst_iterator;
    ++src_iterator;
  }
}

} // namespace kernel
} // namespace threadblock
} // namespace transform
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined) {

  using Shape = cutlass::layout::PitchLinearShape<64, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinearStripminedThreadMap<Shape, kThreads>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator<
    Shape, Element, Layout, 1, ThreadMap
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(57, 35);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(64, 35);
  
  cutlass::HostTensor<int, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::BlockFillSequential(src_tensor.host_data(), src_tensor.capacity());

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]; ++s) {
    for (int c = 0; c < alloc_extent[0]; ++c) {

      Element expected = Element(0);

      if (c < copy_extent[0] && s < copy_extent[1]) {
        expected = src_tensor.at({c, s});
      }
      else {
        expected = oob_value;
      }

      Element got = dst_tensor.at({c, s});
      bool equal = (expected == got);

      EXPECT_EQ(expected, got)
        << "Source:\n" << src_tensor.host_view() << "\n\n"
        << "Destination:\n" << dst_tensor.host_view() << "\n";

      if (!equal) {
        return;
      }
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_128x4) {

  using Shape = cutlass::layout::PitchLinearShape<128, 4>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap, false
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(128, 4);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(128, 4);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::BlockFillSequential(src_tensor.host_data(), src_tensor.capacity());

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]; ++s) {
    for (int c = 0; c < alloc_extent[0]; ++c) {

      Element expected = Element(0);

      if (c < copy_extent[0] && s < copy_extent[1]) {
        expected = src_tensor.at({c, s});
      }
      else {
        expected = oob_value;
      }

      Element got = dst_tensor.at({c, s});
      bool equal = (expected == got);

      EXPECT_EQ(expected, got)
        << "Source:\n" << src_tensor.host_view() << "\n\n"
        << "Destination:\n" << dst_tensor.host_view() << "\n";

      if (!equal) {
        return;
      }
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_128x64) {

  using Shape = cutlass::layout::PitchLinearShape<128, 64>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(128, 64);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(128, 64);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::BlockFillSequential(src_tensor.host_data(), src_tensor.capacity());

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]; ++s) {
    for (int c = 0; c < alloc_extent[0]; ++c) {

      Element expected = Element(0);

      if (c < copy_extent[0] && s < copy_extent[1]) {
        expected = src_tensor.at({c, s});
      }
      else {
        expected = oob_value;
      }

      Element got = dst_tensor.at({c, s});
      bool equal = (expected == got);

      EXPECT_EQ(expected, got)
        << "Source:\n" << src_tensor.host_view() << "\n\n"
        << "Destination:\n" << dst_tensor.host_view() << "\n";

      if (!equal) {
        return;
      }
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_64x64) {

  using Shape = cutlass::layout::PitchLinearShape<64, 64>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(64, 64);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(64, 64);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::BlockFillSequential(src_tensor.host_data(), src_tensor.capacity());

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]; ++s) {
    for (int c = 0; c < alloc_extent[0]; ++c) {

      Element expected = Element(0);

      if (c < copy_extent[0] && s < copy_extent[1]) {
        expected = src_tensor.at({c, s});
      }
      else {
        expected = oob_value;
      }

      Element got = dst_tensor.at({c, s});
      bool equal = (expected == got);

      EXPECT_EQ(expected, got)
        << "Source:\n" << src_tensor.host_view() << "\n\n"
        << "Destination:\n" << dst_tensor.host_view() << "\n";

      if (!equal) {
        return;
      }
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_64x8) {

  using Shape = cutlass::layout::PitchLinearShape<64, 8>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(32, 8);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(64, 8);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::BlockFillSequential(src_tensor.host_data(), src_tensor.capacity());

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]; ++s) {
    for (int c = 0; c < alloc_extent[0]; ++c) {

      Element expected = Element(0);

      if (c < copy_extent[0] && s < copy_extent[1]) {
        expected = src_tensor.at({c, s});
      }
      else {
        expected = oob_value;
      }

      Element got = dst_tensor.at({c, s});
      bool equal = (expected == got);

      EXPECT_EQ(expected, got)
        << "Source:\n" << src_tensor.host_view() << "\n\n"
        << "Destination:\n" << dst_tensor.host_view() << "\n";

      if (!equal) {
        return;
      }
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_64x32_transpose4x4) {

  using Shape = cutlass::layout::PitchLinearShape<64, 8>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap, true
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(64, 32);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(64, 32);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  uint64_t seed = 7;
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::TensorFillRandomUniform(src_tensor.host_view(), seed, 8, -8, 0);

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]/4; ++s) {
    for (int c = 0; c < alloc_extent[0]/4; ++c) {
      for (int s1 = 0; s1 < 4; s1++){
        for(int c1 = 0; c1 < 4; c1++){
          Element expected = Element(0);

          int l_c = c * 4 + c1;
          int l_s = s * 4 + s1;

          int l_tc = c * 4 + s1;
          int l_ts = s * 4 + c1;

          if (l_c < copy_extent[0] && l_s < copy_extent[1]) {
            expected = src_tensor.at({l_c, l_s});
          }
          else {
            expected = oob_value;
          }    

          Element got = dst_tensor.at({l_tc, l_ts});
          bool equal = (expected == got);

          EXPECT_EQ(expected, got)
            << "Source:\n" << src_tensor.host_view() << "\n\n"
            << "Destination:\n" << dst_tensor.host_view() << "\n";

          if (!equal) {
            return;
          }
        }
      }
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_64x29_transpose4x4) {

  using Shape = cutlass::layout::PitchLinearShape<64, 8>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap, true
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(64, 29);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(64, 29);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  uint64_t seed = 7;
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::TensorFillRandomUniform(src_tensor.host_view(), seed, 8, -8, 0);

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]/4; ++s) {
    for (int c = 0; c < alloc_extent[0]/4; ++c) {
      for (int s1 = 0; s1 < 4; s1++){
        for(int c1 = 0; c1 < 4; c1++){
          Element expected = Element(0);

          int l_c = c * 4 + c1;
          int l_s = s * 4 + s1;

          int l_tc = c * 4 + s1;
          int l_ts = s * 4 + c1;

          if (l_c < copy_extent[0] && l_s < copy_extent[1]) {
            expected = src_tensor.at({l_c, l_s});
          }
          else {
            expected = oob_value;
          }    

          Element got = dst_tensor.at({l_tc, l_ts});
          bool equal = (expected == got);

          EXPECT_EQ(expected, got)
            << "Source:\n" << src_tensor.host_view() << "\n\n"
            << "Destination:\n" << dst_tensor.host_view() << "\n";

          if (!equal) {
            return;
          }
        }
      }
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_120x4_transpose4x4) {

  using Shape = cutlass::layout::PitchLinearShape<128, 4>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap, true
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(120, 4);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(120, 4);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  uint64_t seed = 7;
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::TensorFillRandomUniform(src_tensor.host_view(), seed, 8, -8, 0);

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]/4; ++s) {
    for (int c = 0; c < alloc_extent[0]/4; ++c) {
      for (int s1 = 0; s1 < 4; s1++){
        for(int c1 = 0; c1 < 4; c1++){
          Element expected = Element(0);

          int l_c = c * 4 + c1;
          int l_s = s * 4 + s1;

          int l_tc = c * 4 + s1;
          int l_ts = s * 4 + c1;

          if (l_c < copy_extent[0] && l_s < copy_extent[1]) {
            expected = src_tensor.at({l_c, l_s});
          }
          else {
            expected = oob_value;
          }    

          Element got = dst_tensor.at({l_tc, l_ts});
          bool equal = (expected == got);

          EXPECT_EQ(expected, got)
            << "Source:\n" << src_tensor.host_view() << "\n\n"
            << "Destination:\n" << dst_tensor.host_view() << "\n";

          if (!equal) {
            return;
          }
        }
      }
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Transform_threadblock_PredicatedTileIterator, PitchLinear_Stripmined_2dtile_48x29_transpose4x4) {

  using Shape = cutlass::layout::PitchLinearShape<64, 8>;
  using ThreadTileShape = cutlass::layout::PitchLinearShape<4, 4>;
  using Layout = cutlass::layout::PitchLinear;
  using Element = int8_t;
  static int const kThreads = 32;
  
  using ThreadMap = cutlass::transform::PitchLinear2DThreadTileStripminedThreadMap<Shape, kThreads, ThreadTileShape>;

  using Iterator = cutlass::transform::threadblock::PredicatedTileIterator2dThreadTile<
    Shape, Element, Layout, 1, ThreadMap, true
  >;

  cutlass::Coord<2> copy_extent = cutlass::make_Coord(48, 29);
  cutlass::Coord<2> alloc_extent = cutlass::make_Coord(48, 29);
  
  cutlass::HostTensor<int8_t, Layout> src_tensor(alloc_extent);
  cutlass::HostTensor<int8_t, Layout> dst_tensor(alloc_extent);

  Element oob_value = Element(-1);
  uint64_t seed = 7;
  cutlass::reference::host::TensorFill(dst_tensor.host_view(), oob_value);
  cutlass::reference::host::TensorFillRandomUniform(src_tensor.host_view(), seed, 8, -8, 0);

  dst_tensor.sync_device();
  src_tensor.sync_device();

  typename Iterator::Params dst_params(dst_tensor.layout());
  typename Iterator::Params src_params(src_tensor.layout());

  dim3 block(kThreads, 1);
  dim3 grid(1, 1);

  test::transform::threadblock::kernel::copy<Iterator><<< grid, block >>>(
    dst_params,
    dst_tensor.device_data(),
    src_params,
    src_tensor.device_data(),
    copy_extent
  );

  hipError_t result = hipGetLastError();
  EXPECT_EQ(result, hipSuccess) << " - CUDA error: " << hipGetErrorString(result);
  
  dst_tensor.sync_host();

  for (int s = 0; s < alloc_extent[1]/4; ++s) {
    for (int c = 0; c < alloc_extent[0]/4; ++c) {
      for (int s1 = 0; s1 < 4; s1++){
        for(int c1 = 0; c1 < 4; c1++){
          Element expected = Element(0);

          int l_c = c * 4 + c1;
          int l_s = s * 4 + s1;

          int l_tc = c * 4 + s1;
          int l_ts = s * 4 + c1;

          if (l_c < copy_extent[0] && l_s < copy_extent[1]) {
            expected = src_tensor.at({l_c, l_s});
          }
          else {
            expected = oob_value;
          }    

          Element got = dst_tensor.at({l_tc, l_ts});
          bool equal = (expected == got);

          EXPECT_EQ(expected, got)
            << "Source:\n" << src_tensor.host_view() << "\n\n"
            << "Destination:\n" << dst_tensor.host_view() << "\n";
          if (!equal) {
            return;
          }
        }
      }
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
