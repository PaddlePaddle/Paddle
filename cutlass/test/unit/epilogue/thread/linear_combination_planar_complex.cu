#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for thread-level GEMM
*/

#include "../../common/cutlass_unit_test.h"

#include "cutlass/epilogue/thread/linear_combination_planar_complex.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace epilogue {
namespace thread {

using FunctorPlanarComplexF32F32 = cutlass::epilogue::thread::LinearCombinationPlanarComplex<
  float, 
  4,
  float, 
  float>;

__global__ void epilogue_thread_functor_planar_complex_f32_f32(
  float *output_ptr, 
  float const *accum_ptr, 
  float const *source_ptr,
  typename FunctorPlanarComplexF32F32::Params params) {

  FunctorPlanarComplexF32F32 linear_combination_op(params);

  auto accum = *reinterpret_cast<cutlass::ArrayPlanarComplex<float , 4> const *>(accum_ptr);
  auto source = *reinterpret_cast<cutlass::ArrayPlanarComplex<float, 4> const *>(source_ptr);

  *reinterpret_cast<cutlass::ArrayPlanarComplex<float, 4>*>(output_ptr) = linear_combination_op(accum, source);
}

}
}
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Epilogue_thread_linear_combination_planar_complex, f32) {

  using Element = float;
  using ElementOutput = float;
  int const kCount = 4;

  using Functor = cutlass::epilogue::thread::LinearCombinationPlanarComplex<
    ElementOutput, 
    kCount,
    Element, 
    Element>;

  cutlass::complex<Element> alpha(Element(2), Element(1));
  cutlass::complex<Element> beta(Element(1), Element(-1));

  typename Functor::Params params(alpha, beta);

  Functor linear_combination_op(params);

  cutlass::ArrayPlanarComplex<ElementOutput, kCount> source;
  cutlass::ArrayPlanarComplex<Element, kCount> accum;

  // Define arbitrary inputs
  for (int i = 0; i < kCount; ++i) {
    accum.real[i] = Element(i * 2);
    accum.imag[i] = Element((i * 3 % 6) - 3);
    source.real[i] = ElementOutput((i * 7 % 9) - 4);
    source.imag[i] = ElementOutput(((i * 5 + 2) % 9) - 4);
  }

  cutlass::ArrayPlanarComplex<ElementOutput, kCount> destination = linear_combination_op(accum, source);

  // Verify each result
  for (int i = 0; i < kCount; ++i) {
    
    cutlass::complex<Element> expected = alpha * cutlass::complex<Element>(accum.real[i], accum.imag[i]) + 
      beta * cutlass::complex<Element>(Element(source.real[i]), Element(source.imag[i]));

    cutlass::complex<ElementOutput> got(destination.real[i], destination.imag[i]);
    
    EXPECT_TRUE(ElementOutput(expected.real()) == got.real());
    EXPECT_TRUE(ElementOutput(expected.imag()) == got.imag());
    EXPECT_TRUE(expected.real() != Element(0) || expected.imag() != Element(0));
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace epilogue {
namespace thread {

using FunctorPlanarComplexF16F32 = cutlass::epilogue::thread::LinearCombinationPlanarComplex<
  cutlass::half_t, 
  4,
  float, 
  float>;

__global__ void epilogue_thread_functor_planar_complex_f16_f32(
  cutlass::half_t *output_ptr, 
  float const *accum_ptr, 
  cutlass::half_t const *source_ptr,
  typename FunctorPlanarComplexF16F32::Params params,
  int N) {

  FunctorPlanarComplexF16F32 linear_combination_op(params);

  
  auto accum = *reinterpret_cast<cutlass::ArrayPlanarComplex<float , 4> const *>(accum_ptr);   
  auto source = *reinterpret_cast<cutlass::ArrayPlanarComplex<cutlass::half_t , 4> const *>(source_ptr);

  #pragma unroll 1
  for (int n = 0; n < N; ++n) {
    source = linear_combination_op(accum, source);
  }

  *reinterpret_cast<cutlass::ArrayPlanarComplex<cutlass::half_t , 4>*>(output_ptr) = source;
}

}
}
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Epilogue_thread_linear_combination_planar_complex, f16_f32) {

  using Element = float;
  using ElementOutput = cutlass::half_t;
  int const kCount = 4;

  using Functor = cutlass::epilogue::thread::LinearCombinationPlanarComplex<
    ElementOutput, 
    kCount,
    Element, 
    Element>;

  cutlass::complex<Element> alpha(Element(2), Element(1));
  cutlass::complex<Element> beta(Element(1), Element(-1));

  typename Functor::Params params(alpha, beta);

  Functor linear_combination_op(params);

  cutlass::ArrayPlanarComplex<ElementOutput, kCount> source;
  cutlass::ArrayPlanarComplex<Element, kCount> accum;

  // Define arbitrary inputs
  for (int i = 0; i < kCount; ++i) {
    accum.real[i] = Element(i * 2);
    accum.imag[i] = Element((i * 3 % 6) - 3);
    source.real[i] = ElementOutput((i * 7 % 9) - 4);
    source.imag[i] = ElementOutput(((i * 5 + 2) % 9) - 4);
  }

  cutlass::ArrayPlanarComplex<ElementOutput, kCount> destination = linear_combination_op(accum, source);

  // Verify each result
  for (int i = 0; i < kCount; ++i) {
    
    cutlass::complex<Element> expected = alpha * cutlass::complex<Element>(accum.real[i], accum.imag[i]) + 
      beta * cutlass::complex<Element>(Element(source.real[i]), Element(source.imag[i]));

    cutlass::complex<ElementOutput> got(destination.real[i], destination.imag[i]);
    
    EXPECT_TRUE(ElementOutput(expected.real()) == got.real());
    EXPECT_TRUE(ElementOutput(expected.imag()) == got.imag());
    EXPECT_TRUE(expected.real() != Element(0) || expected.imag() != Element(0));
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace epilogue {
namespace thread {

using FunctorPlanarComplexF16F16 = cutlass::epilogue::thread::LinearCombinationPlanarComplex<
  cutlass::half_t, 
  4,
  cutlass::half_t, 
  cutlass::half_t>;

__global__ void epilogue_thread_functor_planar_complex_f16_f16(
  cutlass::half_t *output_ptr, 
  cutlass::half_t const *accum_ptr, 
  cutlass::half_t const *source_ptr,
  typename FunctorPlanarComplexF16F16::Params params,
  int N) {

  FunctorPlanarComplexF16F16 linear_combination_op(params);

  auto accum = *reinterpret_cast<cutlass::ArrayPlanarComplex<cutlass::half_t , 4> const *>(accum_ptr);
  auto source = *reinterpret_cast<cutlass::ArrayPlanarComplex<cutlass::half_t , 4> const *>(source_ptr);

  #pragma unroll 1
  for (int n = 0; n < N; ++n) {
    source = linear_combination_op(accum, source);
  }

  *reinterpret_cast<cutlass::ArrayPlanarComplex<cutlass::half_t , 4>*>(output_ptr) = source;
}

}
}
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Epilogue_thread_linear_combination_planar_complex, f16_f16) {

  using Element = cutlass::half_t;
  using ElementOutput = cutlass::half_t;
  int const kCount = 8;

  using Functor = cutlass::epilogue::thread::LinearCombinationPlanarComplex<
    ElementOutput, 
    kCount,
    Element, 
    Element>;

  cutlass::complex<Element> alpha(Element(2), Element(1));
  cutlass::complex<Element> beta(Element(1), Element(-1));

  typename Functor::Params params(alpha, beta);

  Functor linear_combination_op(params);

  cutlass::ArrayPlanarComplex<ElementOutput, kCount> source;
  cutlass::ArrayPlanarComplex<Element, kCount> accum;

  // Define arbitrary inputs
  for (int i = 0; i < kCount; ++i) {
    accum.real[i] = Element(i * 2);
    accum.imag[i] = Element((i * 3 % 6) - 3);
    source.real[i] = ElementOutput((i * 7 % 9) - 4);
    source.imag[i] = ElementOutput(((i * 5 + 2) % 9) - 4);
  }

  cutlass::ArrayPlanarComplex<ElementOutput, kCount> destination = linear_combination_op(accum, source);

  // Verify each result
  for (int i = 0; i < kCount; ++i) {
    
    cutlass::complex<Element> expected = alpha * cutlass::complex<Element>(accum.real[i], accum.imag[i]) + 
      beta * cutlass::complex<Element>(Element(source.real[i]), Element(source.imag[i]));

    cutlass::complex<ElementOutput> got(destination.real[i], destination.imag[i]);
    
    EXPECT_TRUE(ElementOutput(expected.real()) == got.real());
    EXPECT_TRUE(ElementOutput(expected.imag()) == got.imag());
    EXPECT_TRUE(expected.real() != Element(0) || expected.imag() != Element(0));
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
