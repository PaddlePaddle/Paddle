#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Statically sized array of elements that accommodates all CUTLASS-supported numeric types
           and is safe to use in a union.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/array.h"
#include "cutlass/util/device_memory.h"
#pragma warning( disable : 4800)
/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace core {

/// Each thread clears its array and writes to global memory. No PRMT instructions should
/// be generated if Array<T, N> is a multiple of 32 bits.
template <typename T, int N>
__global__ void test_array_clear(cutlass::Array<T, N> *ptr) {

  cutlass::Array<T, N> storage;

  storage.clear();

  ptr[threadIdx.x] = storage;
}

/// Each thread writes its thread index into the elements of its array and then writes the result
/// to global memory.
template <typename T, int N>
__global__ void test_array_threadid(cutlass::Array<T, N> *ptr) {

  cutlass::Array<T, N> storage;

  CUTLASS_PRAGMA_UNROLL
  for (int i = 0; i < N; ++i) {
    storage.at(i) = T(int(threadIdx.x));
  }

  ptr[threadIdx.x] = storage;
}

/// Each thread writes its thread index into the elements of its array and then writes the result
/// to global memory.
template <typename T, int N>
__global__ void test_array_sequence(cutlass::Array<T, N> *ptr) {

  cutlass::Array<T, N> storage;

  CUTLASS_PRAGMA_UNROLL
  for (int i = 0; i < N; ++i) {
    storage.at(i) = T(i);
  }

  ptr[threadIdx.x] = storage;
}

} // namespace core
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int N>
class TestArray {
public:

  //
  // Data members
  //

  /// Number of threads
  int const kThreads = 32;

  typedef cutlass::Array<T, N> ArrayTy;


  //
  // Methods
  //

  /// Ctor
  TestArray() {

  }

  /// Runs the test
  void run() {

    /// Device memory containing output
    cutlass::device_memory::allocation< ArrayTy > output(kThreads);
    std::vector< ArrayTy > output_host(kThreads);

    dim3 grid(1,1);
    dim3 block(kThreads, 1, 1);

    test::core::test_array_clear<<< grid, block >>>(output.get());

    hipError_t result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "CUDA error: " << hipGetErrorString(result);

    //
    // Verify contains all zeros
    //

    cutlass::device_memory::copy_to_host(output_host.data(), output.get(), kThreads);

    result = hipGetLastError();
    ASSERT_EQ(result, hipSuccess) << "CUDA error: " << hipGetErrorString(result);

    char const *ptr_host = reinterpret_cast<char const *>(output_host.data());
    for (int i = 0; i < sizeof(ArrayTy) * kThreads; ++i) {
      EXPECT_FALSE(ptr_host[i]);
    }

    //
    // Verify each element contains the low bits of the thread Id
    //

    test::core::test_array_threadid<<< grid, block >>>(output.get());

    result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "CUDA error: " << hipGetErrorString(result);

    cutlass::device_memory::copy_to_host(output_host.data(), output.get(), kThreads);

    result = hipGetLastError();
    ASSERT_EQ(result, hipSuccess) << "CUDA error: " << hipGetErrorString(result);

    for (int i = 0; i < kThreads; ++i) {
      T tid = T(i);

      ArrayTy thread = output_host.at(i);

      // Element-wise access
      for (int j = 0; j < N; ++j) {
        EXPECT_TRUE(tid == thread[j]);
      }

      // Iterator access
      for (auto it = thread.begin(); it != thread.end(); ++it) {
        EXPECT_TRUE(tid == *it);
      }

      // Range-based for
      for (auto const & x : thread) {
        EXPECT_TRUE(tid == x);
      }
    }

    //
    // Verify each element
    //

    test::core::test_array_sequence<<< grid, block >>>(output.get());

    result = hipDeviceSynchronize();
    ASSERT_EQ(result, hipSuccess) << "CUDA error: " << hipGetErrorString(result);

    cutlass::device_memory::copy_to_host(output_host.data(), output.get(), kThreads);

    result = hipGetLastError();
    ASSERT_EQ(result, hipSuccess) << "CUDA error: " << hipGetErrorString(result);

    for (int i = 0; i < kThreads; ++i) {

      ArrayTy thread = output_host.at(i);

      // Element-wise access
      for (int j = 0; j < N; ++j) {
        T got = T(j);
        EXPECT_TRUE(got == thread[j]);
      }

      // Iterator access
      int j = 0;
      for (auto it = thread.begin(); it != thread.end(); ++it, ++j) {
        T got = T(j);
        EXPECT_TRUE(got == *it);
      }

      // Range-based for
      j = 0;
      for (auto const & x : thread) {
        T got = T(j);
        EXPECT_TRUE(got == x);
        ++j;
      }
    }
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////
TEST(Array, Int8x16) {
  TestArray<int8_t, 16>().run();
}

TEST(Array, Int32x4) {
  TestArray<int, 4>().run();
}

#if __CUDA_ARCH__ >= 520
TEST(Array, Float16x8) {
  TestArray<cutlass::half_t, 8>().run();
}
#endif

TEST(Array, FloatBF16x8) {
  TestArray<cutlass::bfloat16_t, 8>().run();
}

TEST(Array, FloatTF32x4) {
  TestArray<cutlass::tfloat32_t, 4>().run();
}

TEST(Array, Float32x4) {
  TestArray<float, 4>().run();
}

TEST(Array, Int4x32) {
  TestArray<cutlass::int4b_t, 32>().run();
}

TEST(Array, Uint4x32) {
  TestArray<cutlass::uint4b_t, 32>().run();
}

TEST(Array, Bin1x128) {
  TestArray<cutlass::bin1_t, 128>().run();
}

/////////////////////////////////////////////////////////////////////////////////////////////////
