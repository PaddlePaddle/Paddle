#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Statically sized array of elements that accommodates all CUTLASS-supported numeric types
           and is safe to use in a union.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/array.h"
#include "cutlass/numeric_types.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/util/device_memory.h"

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Host
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(tfloat32_t, host_conversion) {
  for (int i = -1024; i < 1024; ++i) {
    float f = static_cast<float>(i);

    cutlass::tfloat32_t x = static_cast<cutlass::tfloat32_t>(i);
    cutlass::tfloat32_t y = static_cast<cutlass::tfloat32_t>(f);

    EXPECT_TRUE(static_cast<int>(x) == i);
    EXPECT_TRUE(static_cast<float>(y) == f);
  }

  // Try out default-ctor (zero initialization of primitive proxy type)
  EXPECT_TRUE(cutlass::tfloat32_t() == 0.0_tf32);

  // Try out user-defined literals
  EXPECT_TRUE(cutlass::tfloat32_t(7) == 7_tf32);
  EXPECT_TRUE(7 == static_cast<int>(7_tf32));
}

TEST(tfloat32_t, host_arithmetic) {

  for (int i = -100; i < 100; ++i) {
    for (int j = -100; j < 100; ++j) {

      cutlass::tfloat32_t x = static_cast<cutlass::tfloat32_t>(i);
      cutlass::tfloat32_t y = static_cast<cutlass::tfloat32_t>(j);

      EXPECT_TRUE(static_cast<int>(x + y) == (i + j));
    }
  }
}

TEST(tfloat32_t, host_round_nearest) {
  
  struct {
    uint32_t f32_bits;
    uint32_t expected;
  } tests[] = {
    {0x40000000, 0x40000000},  // M=0, R=0, S=0 => rtz
    {0x40001000, 0x40000000},  // M=0, R=1, S=0 => rtz
    {0x40000001, 0x40000000},  // M=0, R=0, S=1 => rtz
    {0x40001001, 0x40002000},  // M=0, R=1, S=1 => +inf
    {0x40002000, 0x40002000},  // M=1, R=0, S=0 => rtz
    {0x40002001, 0x40002000},  // M=1, R=0, S=1 => rtz
    {0x40003000, 0x40004000},  // M=1, R=1, S=0 => +inf
    {0x40003001, 0x40004000},  // M=1, R=1, S=1 => +inf
    {0x7f800000, 0x7f800000},  // +inf
    {0xff800000, 0xff800000},  // -inf
    {0x7fffffff, 0x7fffffff},  // canonical NaN to canonical NaN
    {0x7f800001, 0x7fffffff},  // NaN to canonical NaN
    {0xff800001, 0x7fffffff},  // NaN to canonical NaN
    {0, 0}
  };
  
  bool running = true;
  for (int i = 0; running; ++i) {

    float f32 = reinterpret_cast<float const &>(tests[i].f32_bits);

    cutlass::NumericConverter<
      cutlass::tfloat32_t, 
      float, 
      cutlass::FloatRoundStyle::round_to_nearest> converter;

    cutlass::tfloat32_t tf32 = converter(f32);
    
    // note, we must explicitly truncate the low-order bits since they are not defined in TF32.
    if (cutlass::isfinite(tf32)) {
      tf32.storage &= 0xffffe000;
    }

    bool passed = (tests[i].expected == tf32.raw());
    
    EXPECT_TRUE(passed)
      << "Error - convert(f32: 0x" << std::hex << tests[i].f32_bits 
      << ") -> 0x" << std::hex << tests[i].expected << "\ngot: 0x" << std::hex << tf32.raw();

    if (!tests[i].f32_bits) {
      running = false;
    }
  }
}

namespace test {
namespace core {

__global__ void convert_tf32_half_ulp(cutlass::tfloat32_t *out, float const *in) {

  cutlass::NumericConverter<
    cutlass::tfloat32_t, 
    float,
    cutlass::FloatRoundStyle::round_half_ulp_truncate> convert;

  *out = convert(*in);
}

}
}


TEST(tfloat32_t, host_round_half_ulp) {
  
  struct {
    uint32_t f32_bits;
    uint32_t expected;
  } tests[] = {
    {0x40001fff, 0x40002000},
    {0x40000000, 0x40000000},  // M=0, R=0, S=0 => rtz
    {0x40001000, 0x40002000},  // M=0, R=1, S=0 => rtz  - this difers from RNE
    {0x40000001, 0x40000000},  // M=0, R=0, S=1 => rtz
    {0x40001001, 0x40002000},  // M=0, R=1, S=1 => +inf
    {0x40002000, 0x40002000},  // M=1, R=0, S=0 => rtz
    {0x40002001, 0x40002000},  // M=1, R=0, S=1 => rtz
    {0x40003000, 0x40004000},  // M=1, R=1, S=0 => +inf
    {0x40003001, 0x40004000},  // M=1, R=1, S=1 => +inf
    {0x7f800000, 0x7f800000},  // +inf
    {0xff800000, 0xff800000},  // -inf
    {0x7fffffff, 0x7fffffff},  // canonical NaN to canonical NaN
    {0x7f800001, 0x7f800001},  // NaN to NaN
    {0xff800001, 0xff800001},  // NaN to NaN
    {0, 0}
  };

  cutlass::NumericConverter<
    cutlass::tfloat32_t, 
    float,
    cutlass::FloatRoundStyle::round_half_ulp_truncate> convert;
  
  bool running = true;
  for (int i = 0; running; ++i) {

    float f32 = reinterpret_cast<float const &>(tests[i].f32_bits);

    cutlass::tfloat32_t tf32 = convert(f32);

    // note, for this test, we must explicitly truncate the low-order bits since they are not 
    // defined in TF32.
    if (cutlass::isfinite(tf32)) {
      tf32.storage &= 0xffffe000;
    }

    bool passed = (tests[i].expected == tf32.raw());
    
    EXPECT_TRUE(passed)
      << "Error - convert(f32: 0x" << std::hex << tests[i].f32_bits 
      << ") -> 0x" << std::hex << tests[i].expected << "\ngot: 0x" << std::hex << tf32.raw();

    if (!tests[i].f32_bits) {
      running = false;
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Device
//
/////////////////////////////////////////////////////////////////////////////////////////////////
