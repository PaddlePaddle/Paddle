#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for functional operators.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/functional.h"
#include "cutlass/core_io.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/util/host_tensor.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace core {
namespace kernel {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Conversion template
template <typename Element, typename Operator>
__global__ void unary_operator(Element *d, Element const *a) {

  Operator op;

  *d = op(*a);
}

/// Conversion template
template <typename Element, typename Operator>
__global__ void binary_operator(Element *d, Element const *a, Element const *b, int Iterations = 1) {

  Operator op;

  Element a_x = *a;
  Element b_x = *b;

  CUTLASS_PRAGMA_NO_UNROLL
  for (int i = 0; i < Iterations; ++i) {
    b_x = op(a_x, b_x);
  }
  
  *d = b_x;
}

/// Conversion template
template <typename Element, typename Operator>
__global__ void trinary_operator(
  Element *d, 
  Element const *a, 
  Element const *b, 
  Element const *c, 
  int Iterations = 1) {

  Operator op;

  Element a_x = a[blockIdx.x];
  Element b_x = b[blockIdx.x];
  Element c_x = c[blockIdx.x];

  CUTLASS_PRAGMA_NO_UNROLL
  for (int i = 0; i < Iterations; ++i) {
    c_x = op(a_x, b_x, c_x);
  }
  
  d[blockIdx.x] = c_x;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace kernel
} // namespace core
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

template <int kN>
void Functional_plus_f16xN() {

  using Element = cutlass::Array<cutlass::half_t, kN>;
  using Operator = cutlass::plus<Element>;

  using Tensor = cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor>;

  Tensor D({1, kN});
  Tensor A({1, kN});
  Tensor B({1, kN});
  Tensor C({1, kN});

  for (int i = 0; i < kN; ++i) {
    A.host_data()[i] = cutlass::half_t((i * 2 + 1) % 5);
    B.host_data()[i] = cutlass::half_t((i * 4 + 8) % 7);
    D.host_data()[i] = cutlass::half_t(0);
  }

  D.sync_device();
  A.sync_device();
  B.sync_device();

  test::core::kernel::binary_operator<Element, Operator><<< dim3(1,1), dim3(1,1) >>>(
    reinterpret_cast<Element *>(D.device_data()),
    reinterpret_cast<Element const *>(A.device_data()),
    reinterpret_cast<Element const *>(B.device_data())
  );

  D.sync_host();

  bool some_d_nonzero = false;

  for (int i = 0; i < kN; ++i) {
    float a = float(A.host_data()[i]);
    float b = float(B.host_data()[i]);
    float d = float(D.host_data()[i]);

    EXPECT_TRUE(d == (a + b));

    if (d != 0) {
      some_d_nonzero = true;
    }
  }

  EXPECT_TRUE(some_d_nonzero);
}

TEST(Functional, plus_f16x16) {
  Functional_plus_f16xN<16>();
}

TEST(Functional, plus_f16x17) {
  Functional_plus_f16xN<17>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <int kN>
void Functional_minus_f16xN() {

  using Element = cutlass::Array<cutlass::half_t, kN>;
  using Operator = cutlass::minus<Element>;

  using Tensor = cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor>;

  Tensor D({1, kN});
  Tensor A({1, kN});
  Tensor B({1, kN});
  Tensor C({1, kN});

  for (int i = 0; i < kN; ++i) {
    A.host_data()[i] = cutlass::half_t((i * 2 + 1) % 5);
    B.host_data()[i] = cutlass::half_t((i * 4 + 8) % 7);
    D.host_data()[i] = cutlass::half_t(0);
  }

  D.sync_device();
  A.sync_device();
  B.sync_device();

  test::core::kernel::binary_operator<Element, Operator><<< dim3(1,1), dim3(1,1) >>>(
    reinterpret_cast<Element *>(D.device_data()),
    reinterpret_cast<Element const *>(A.device_data()),
    reinterpret_cast<Element const *>(B.device_data())
  );

  D.sync_host();

  bool some_d_nonzero = false;

  for (int i = 0; i < kN; ++i) {
    float a = float(A.host_data()[i]);
    float b = float(B.host_data()[i]);
    float d = float(D.host_data()[i]);

    EXPECT_TRUE(d == (a - b));

    if (d != 0) {
      some_d_nonzero = true;
    }
  }

  EXPECT_TRUE(some_d_nonzero);
}

TEST(Functional, minus_f16x16) {
  Functional_minus_f16xN<16>();
}

TEST(Functional, minus_f16x17) {
  Functional_minus_f16xN<17>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <int kN>
void Functional_multiplies_f16xN() {

  using Element = cutlass::Array<cutlass::half_t, kN>;
  using Operator = cutlass::multiplies<Element>;

  using Tensor = cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor>;

  Tensor D({1, kN});
  Tensor A({1, kN});
  Tensor B({1, kN});
  Tensor C({1, kN});

  for (int i = 0; i < kN; ++i) {
    A.host_data()[i] = cutlass::half_t((i * 2 + 1) % 5);
    B.host_data()[i] = cutlass::half_t((i * 4 + 8) % 7);
    D.host_data()[i] = cutlass::half_t(0);
  }

  D.sync_device();
  A.sync_device();
  B.sync_device();

  test::core::kernel::binary_operator<Element, Operator><<< dim3(1,1), dim3(1,1) >>>(
    reinterpret_cast<Element *>(D.device_data()),
    reinterpret_cast<Element const *>(A.device_data()),
    reinterpret_cast<Element const *>(B.device_data())
  );

  D.sync_host();

  bool some_d_nonzero = false;

  for (int i = 0; i < kN; ++i) {
    float a = float(A.host_data()[i]);
    float b = float(B.host_data()[i]);
    float d = float(D.host_data()[i]);

    EXPECT_TRUE(d == (a * b));

    if (d != 0) {
      some_d_nonzero = true;
    }
  }

  EXPECT_TRUE(some_d_nonzero);
}

TEST(Functional, multiplies_f16x16) {

  Functional_multiplies_f16xN<16>();
}

TEST(Functional, multiplies_f16x17) {

  Functional_multiplies_f16xN<17>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <int kN>
void Functional_divides_f16xN() {

  using Element = cutlass::Array<cutlass::half_t, kN>;
  using Operator = cutlass::divides<Element>;

  using Tensor = cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor>;

  Tensor D({1, kN});
  Tensor A({1, kN});
  Tensor B({1, kN});
  Tensor C({1, kN});

  for (int i = 0; i < kN; ++i) {
    A.host_data()[i] = cutlass::half_t((i * 2 + 1) % 5);
    B.host_data()[i] = cutlass::half_t((i * 4 + 8) % 7);
    D.host_data()[i] = cutlass::half_t(0);
  }

  D.sync_device();
  A.sync_device();
  B.sync_device();

  test::core::kernel::binary_operator<Element, Operator><<< dim3(1,1), dim3(1,1) >>>(
    reinterpret_cast<Element *>(D.device_data()),
    reinterpret_cast<Element const *>(A.device_data()),
    reinterpret_cast<Element const *>(B.device_data())
  );

  D.sync_host();

  bool some_d_nonzero = false;

  for (int i = 0; i < kN; ++i) {
    float a = float(A.host_data()[i]);
    float b = float(B.host_data()[i]);
    float d = float(D.host_data()[i]);

    float expected = a / b;

    float const kThreshold = 0.0005f;

    if (std::isnan(expected)) {
      EXPECT_TRUE(std::isnan(d));
    }
    else if (std::isinf(expected)) {
      EXPECT_TRUE(std::isinf(d));
    }
    else {
      EXPECT_TRUE(std::abs(d - expected) < kThreshold)
        << "Got: " << d << " = " << a << " / " << b << ", expected: " << (a / b); 
    }

    if (d != 0) {
      some_d_nonzero = true;
    }
  }

  EXPECT_TRUE(some_d_nonzero);
}

TEST(Functional, divides_f16x16) {

  Functional_divides_f16xN<16>();
}

TEST(Functional, divides_f16x17) {

  Functional_divides_f16xN<17>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int kN>
void Functional_multiply_add_TxN() {

  using Element = cutlass::Array<T, kN>;
  using Operator = cutlass::multiply_add<Element>;

  using Tensor = cutlass::HostTensor<T, cutlass::layout::RowMajor>;

  Tensor D({1, kN});
  Tensor A({1, kN});
  Tensor B({1, kN});
  Tensor C({1, kN});

  for (int i = 0; i < kN; ++i) {
    A.host_data()[i] = T((i * 2 + 1) % 5);
    B.host_data()[i] = T((i * 4 + 8) % 7);
    C.host_data()[i] = T((i * 3 + 11) % 11);
    D.host_data()[i] = T(0);
  }

  D.sync_device();
  A.sync_device();
  B.sync_device();
  C.sync_device();

  test::core::kernel::trinary_operator<Element, Operator><<< dim3(1,1), dim3(1,1) >>>(
    reinterpret_cast<Element *>(D.device_data()),
    reinterpret_cast<Element const *>(A.device_data()),
    reinterpret_cast<Element const *>(B.device_data()),
    reinterpret_cast<Element const *>(C.device_data())
  );

  D.sync_host();

  bool some_d_nonzero = false;

  for (int i = 0; i < kN; ++i) {
    float a = float(A.host_data()[i]);
    float b = float(B.host_data()[i]);
    float c = float(C.host_data()[i]);
    float d = float(D.host_data()[i]);

    EXPECT_TRUE(d == (a * b + c));

    if (d != 0) {
      some_d_nonzero = true;
    }
  }

  EXPECT_TRUE(some_d_nonzero);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Functional, multiply_add_f16x16) {
  Functional_multiply_add_TxN<cutlass::half_t, 16>();
}

TEST(Functional, multiply_add_f16x17) {
  Functional_multiply_add_TxN<cutlass::half_t, 17>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Functional, multiply_add_bf16x16) {
  Functional_multiply_add_TxN<cutlass::bfloat16_t, 16>();
}

TEST(Functional, multiply_add_bf16x17) {
  Functional_multiply_add_TxN<cutlass::bfloat16_t, 17>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
cutlass::Quaternion<T> random_quaternion(int range) {
  return cutlass::Quaternion<T>{
    T((rand() % range * 2) - range),
    T((rand() % range * 2) - range),
    T((rand() % range * 2) - range),
    T((rand() % range * 2) - range)
  };
}

template <typename T>
void Functional_multiply_add_QuaternionT() {

  using Element = cutlass::Quaternion<T>;
  using Operator = cutlass::multiply_add<Element, Element, Element>;
  using HostTensor = cutlass::HostTensor<Element, cutlass::layout::RowMajor>;

  int const kM = 128;
  int const kRange = 8;

  HostTensor A({kM, 1});
  HostTensor B({kM, 1});
  HostTensor C({kM, 1});
  HostTensor D({kM, 1});

  srand(2021);

  for (int m = 0; m < kM; ++m) {
    A.at({m, 0}) = random_quaternion<T>(kRange);
    B.at({m, 0}) = random_quaternion<T>(kRange);
    C.at({m, 0}) = random_quaternion<T>(kRange);
  }

  A.sync_device();
  B.sync_device();
  C.sync_device();
  D.sync_device();

  test::core::kernel::trinary_operator<Element, Operator><<< dim3(kM,1), dim3(1,1) >>>(
    D.device_data(),
    A.device_data(),
    B.device_data(),
    C.device_data()
  );

  D.sync_host();
  
  for (int m = 0; m < kM; ++m) {

    Element a = A.at({m, 0});
    Element b = B.at({m, 0});
    Element c = C.at({m, 0});
    Element got = D.at({m, 0});
    Element expected = a * b + c;

    EXPECT_TRUE(got == expected);
  }
}

TEST(Functional, multiply_add_quaternion_f32) {
  Functional_multiply_add_QuaternionT<float>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////
