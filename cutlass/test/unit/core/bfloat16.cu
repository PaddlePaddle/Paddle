#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Statically sized array of elements that accommodates all CUTLASS-supported numeric types
           and is safe to use in a union.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/array.h"
#include "cutlass/core_io.h"
#include "cutlass/numeric_types.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/layout/matrix.h"

#include "cutlass/util/device_memory.h"
#include "cutlass/util/host_tensor.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void convert_bf16_f32(cutlass::bfloat16_t *output, float const *input, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) {
    output[tid] = static_cast<cutlass::bfloat16_t>(input[tid]);
  }
}

__global__ void convert_and_pack_bf16(cutlass::bfloat16_t *output, float const *input, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid * 2 < N) {

    cutlass::NumericArrayConverter<cutlass::bfloat16_t, float, 2> convert;

    cutlass::Array<cutlass::bfloat16_t, 2> *dst_ptr = 
      reinterpret_cast<cutlass::Array<cutlass::bfloat16_t, 2> *>(output + tid * 2);

    cutlass::Array<float, 2> const *src_ptr = 
      reinterpret_cast<cutlass::Array<float, 2> const *>(input + tid * 2);

    *dst_ptr = convert(*src_ptr);
  } 
}

TEST(bfloat16_t, device_conversion) {
  using T = cutlass::bfloat16_t;
  using S = float;

  int const N = 256;

  cutlass::HostTensor<T, cutlass::layout::RowMajor> destination({N, 1});
  cutlass::HostTensor<S, cutlass::layout::RowMajor> source({N, 1});

  for (int i = 0; i < N; ++i) {
    source.at({i, 0}) = float(i - 128);
    destination.at({i, 0}) = T(0);
  }

  source.sync_device();
  destination.sync_device();

  convert_bf16_f32<<< dim3(1,1), dim3(N, 1) >>>(destination.device_data(), source.device_data(), N);

  ASSERT_EQ(hipGetLastError(), hipSuccess) << "Kernel launch error.";

  destination.sync_host();

  int errors = 0;
  for (int i = 0; i < N; ++i) {
    T got = destination.at({i, 0});
    S expected = source.at({i, 0});

    if (S(got) != expected) {
      ++errors;
      if (errors < 10) {
        std::cerr << "Basic conversion error - [" << i << "] - got " << got << ", expected " << expected << "\n";
      }
    }

    destination.at({i, 0}) = T(0);
  }

  destination.sync_device();
  
  convert_and_pack_bf16<<< dim3(1,1), dim3(N, 1) >>>(destination.device_data(), source.device_data(), N);
  
  ASSERT_EQ(hipGetLastError(), hipSuccess) << "Kernel launch error.";
  
  destination.sync_host();

  for (int i = 0; i < N; ++i) {
    T got = destination.at({i, 0});
    S expected = source.at({i, 0});

    if (S(got) != expected) {
      ++errors;
      if (errors < 10) {
        std::cerr << "Convert and pack error - [" << i << "] - got " << got << ", expected " << expected << "\n";
      }
    }
  }

  EXPECT_EQ(errors, 0);
}


/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Host
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(bfloat16_t, host_conversion) {
  for (int i = -128; i < 128; ++i) {
    float f = static_cast<float>(i);

    cutlass::bfloat16_t x = static_cast<cutlass::bfloat16_t>(i);
    cutlass::bfloat16_t y = static_cast<cutlass::bfloat16_t>(f);

    EXPECT_TRUE(static_cast<int>(x) == i);
    EXPECT_TRUE(static_cast<float>(y) == f);
  }

  // Try out default-ctor (zero initialization of primitive proxy type)
  EXPECT_TRUE(cutlass::bfloat16_t() == 0.0_bf16);

  // Try out user-defined literals
  EXPECT_TRUE(cutlass::bfloat16_t(7) == 7_bf16);
  EXPECT_TRUE(7 == static_cast<int>(7_bf16));
}

TEST(bfloat16_t, host_arithmetic) {

  for (int i = -100; i < 100; ++i) {
    for (int j = -100; j < 100; ++j) {

      cutlass::bfloat16_t x = static_cast<cutlass::bfloat16_t>(i);
      cutlass::bfloat16_t y = static_cast<cutlass::bfloat16_t>(j);

      EXPECT_TRUE(static_cast<int>(x + y) == (i + j));
    }
  }
}

TEST(bfloat16_t, host_round) {
  
  struct {
    uint32_t f32_bits;
    uint16_t expected;
  } tests[] = {
    {0x40040000, 0x4004},  // M=0, R=0, S=0 => rtz
    {0x40048000, 0x4004},  // M=0, R=1, S=0 => rtz
    {0x40040001, 0x4004},  // M=0, R=1, S=1 => +inf
    {0x4004c000, 0x4005},  // M=0, R=1, S=1 => +inf
    {0x4004a000, 0x4005},  // M=0, R=1, S=1 => +inf
    {0x40050000, 0x4005},  // M=1, R=0, S=0 => rtz
    {0x40054000, 0x4005},  // M=1, R=0, S=1 => rtz
    {0x40058000, 0x4006},  // M=1, R=1, S=0 => +inf
    {0x40058001, 0x4006},  // M=1, R=1, S=1 => +inf
    {0x7f800000, 0x7f80},  // +inf
    {0xff800000, 0xff80},  // -inf
    {0x7fffffff, 0x7fff},  // canonical NaN
    {0x7ff00001, 0x7fff},  // NaN -> canonical NaN
    {0xfff00010, 0x7fff},  // Nan -> canonical NaN
    {0, 0}
  };
  
  bool running = true;
  for (int i = 0; running; ++i) {

    float f32 = reinterpret_cast<float const &>(tests[i].f32_bits);

    cutlass::bfloat16_t bf16 = cutlass::bfloat16_t(f32);

    bool passed = (tests[i].expected == bf16.raw());
    
    EXPECT_TRUE(passed)
      << "Error - convert(f32: 0x" << std::hex << tests[i].f32_bits 
      << ") -> 0x" << std::hex << tests[i].expected << "\ngot: 0x" << std::hex << bf16.raw();

    if (!tests[i].f32_bits) {
      running = false;
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Device
//
/////////////////////////////////////////////////////////////////////////////////////////////////
