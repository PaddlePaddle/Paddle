#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief CUTLASS host-device template for complex numbers supporting all CUTLASS numeric types.
*/

// Standard Library's std::complex<T> used for reference checking
#include <complex>

#include "../common/cutlass_unit_test.h"

#include "cutlass/complex.h"
#include "cutlass/constants.h"
#include "cutlass/numeric_conversion.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, f64_to_f32_conversion) {

  cutlass::complex<double> source = {1.5, -1.25};

  cutlass::complex<float> dest = cutlass::complex<float>(source); // explicit conversion

  EXPECT_TRUE(source.real() == 1.5 && source.imag() == -1.25 && 
    dest.real() == 1.5f && dest.imag() == -1.25f);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, f32_to_f64_conversion) {

  cutlass::complex<float> source = {-1.5f, 1.25f};

  cutlass::complex<double> dest = source;  // implicit conversion

  EXPECT_TRUE(source.real() == -1.5f && source.imag() == 1.25f && 
    dest.real() == -1.5 && dest.imag() == 1.25);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, s32_to_f64_conversion) {

  cutlass::complex<int> source = {-2, 1};

  cutlass::complex<double> dest = source;  // implicit conversion

  EXPECT_TRUE(source.real() == -2 && source.imag() == 1 && 
    dest.real() == -2 && dest.imag() == 1);
}


/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, f16_to_f32_conversion) {

  cutlass::complex<cutlass::half_t> source = {1.5_hf, -1.25_hf};

  cutlass::complex<float> dest = cutlass::complex<float>(source); // explicit conversion

  EXPECT_TRUE(source.real() == 1.5_hf && source.imag() == -1.25_hf && 
    dest.real() == 1.5f && dest.imag() == -1.25f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, exp_f32) {
  
  cutlass::complex<float> Z[] = {
    {1, 1},
    {2   ,  cutlass::constants::pi<float>()/2.0f   },
    {0.5f,  cutlass::constants::pi<float>()        },
    {0.25f,  cutlass::constants::pi<float>()*3/4.0f },
    {0, 0},
  };

  cutlass::complex<double> Expected[] = {
    {1.4686939399158851, 2.2873552871788423}, 
    {4.524491950137825e-16, 7.38905609893065},
    {-1.6487212707001282, 2.019101226849069e-16}, 
    {-0.9079430793557842, 0.9079430793557843},
    {1, 0}
  };

  double tolerance = 0.00001;

  for (int i = 0; cutlass::real(Z[i]); ++i) {
    double e_r = cutlass::real(Expected[i]);
    double e_i = cutlass::real(Expected[i]);

    cutlass::complex<float> got = cutlass::exp(Z[i]);
    float g_r = cutlass::real(got);
    float g_i = cutlass::real(got);

    EXPECT_TRUE(
      std::abs(g_r - e_r) < tolerance && std::abs(g_i - e_i) < tolerance
    ) << "Expected(" << Expected[i] << "), Got(" << got << ")";
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {

  /// Thorough testing for basic complex math operators. Uses std::complex as a reference.
  template <typename T, int N, int M>
  struct ComplexOperators {
    ComplexOperators() {
      for (int ar = -N; ar <= N; ++ar) {
        for (int ai = -N; ai <= N; ++ai) {
          for (int br = -N; br <= N; ++br) {
            for (int bi = -N; bi <= N; ++bi) {

              cutlass::complex<T> Ae(T(ar) / T(M), T(ai) / T(M));
              cutlass::complex<T> Be(T(br) / T(M), T(bi) / T(M));

              std::complex<T> Ar(T(ar) / T(M), T(ai) / T(M));
              std::complex<T> Br(T(br) / T(M), T(bi) / T(M));

              cutlass::complex<T> add_e = Ae + Be;
              cutlass::complex<T> sub_e = Ae - Be;
              cutlass::complex<T> mul_e = Ae * Be;

              std::complex<T> add_r = (Ar + Br);
              std::complex<T> sub_r = (Ar - Br);
              std::complex<T> mul_r = (Ar * Br);

              EXPECT_EQ(real(add_e), real(add_r));
              EXPECT_EQ(imag(add_e), imag(add_r));

              EXPECT_EQ(real(sub_e), real(sub_r));
              EXPECT_EQ(imag(sub_e), imag(sub_r));

              EXPECT_EQ(real(mul_e), real(mul_r));
              EXPECT_EQ(imag(mul_e), imag(mul_r));

              if (!(br == 0 && bi == 0)) {

                cutlass::complex<T> div_e = Ae / Be;
                std::complex<T> div_r = Ar / Br;

                T const kRange = T(0.001);

                EXPECT_NEAR(real(div_e), real(div_r), kRange);
                EXPECT_NEAR(imag(div_e), imag(div_r), kRange);
              }
            }
          }
        }
      }
    }
  };
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, host_float) {
  test::ComplexOperators<float, 32, 8> test;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, host_double) {
  test::ComplexOperators<double, 32, 8> test;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
