#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for conversion operators.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/numeric_conversion.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/util/host_tensor.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace core {
namespace kernel {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Conversion template
template <typename Destination, typename Source, int Count>
__global__ void convert(
  cutlass::Array<Destination, Count> *destination, 
  cutlass::Array<Source, Count> const *source) {

  cutlass::NumericArrayConverter<Destination, Source, Count> convert;

  *destination = convert(*source);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace kernel
} // namespace core
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32_to_f16_rn) {

  int const kN = 1;
  using Source = float;
  using Destination = cutlass::half_t;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<float, cutlass::layout::RowMajor> source({1, kN});

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = float(i);
  }

  source.sync_device();

  test::core::kernel::convert<Destination, Source, 1><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, 1> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, 1> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination.host_data()[i]) == source.host_data()[i]);
  }
}

TEST(NumericConversion, f32x8_to_f16x8_rn) {

  int const kN = 8;
  using Source = float;
  using Destination = cutlass::half_t;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<Destination, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<Source, cutlass::layout::RowMajor> source({1, kN});

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = float(i);
  }

  source.sync_device();

  test::core::kernel::convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination.host_data()[i]) == source.host_data()[i]);
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f16_to_f32_rn) {
  
  int const kN = 1;
  using Source = cutlass::half_t;
  using Destination = float;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<float, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> source({1, kN});

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = Source(i);
  }

  source.sync_device();

  test::core::kernel::convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination.host_data()[i]) == float(source.host_data()[i]));
  }
}

TEST(NumericConversion, f16x8_to_f32x8_rn) {

  int const kN = 8;
  using Source = cutlass::half_t;
  using Destination = float;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<float, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<cutlass::half_t, cutlass::layout::RowMajor> source({1, kN});

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = float(i);
  }

  source.sync_device();

  test::core::kernel::convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination.host_data()[i]) == float(source.host_data()[i]));
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
