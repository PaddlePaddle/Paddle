#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for the CUTLASS Quaternion template class.
*/

#include "../common/cutlass_unit_test.h"
#include "cutlass/cutlass.h"
#include "cutlass/core_io.h"
#include "cutlass/quaternion.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/constants.h"
  
/////////////////////////////////////////////////////////////////////////////////////////////////

static float const half_pi = cutlass::constants::half_pi<float>();

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, add_f32) {

  cutlass::Quaternion<float> q0(1, 1, 1, 1);
  cutlass::Quaternion<float> q1(0, 0, 0, 2);

  cutlass::Quaternion<float> q2 = q0 + q1;

  EXPECT_TRUE(
    q2.x() == 1 &&
    q2.y() == 1 &&
    q2.z() == 1 &&
    q2.w() == 3
  );
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, rotation) {

  cutlass::Matrix3x1<float> x(1.0f, 0.0f, 0.0f);
  cutlass::Quaternion<float> q = cutlass::Quaternion<float>::rotation(0, 0, 1, half_pi) * 2.0f;
  cutlass::Matrix3x1<float> v = q.rotate(x);

  float epsilon = 0.001f;

  EXPECT_TRUE(
    std::abs(v.at(0)) < epsilon &&
    std::abs(v.at(1)) > (1 - epsilon) &&
    std::abs(v.at(2)) < epsilon
  );
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, rotation_inv) {
  
  cutlass::Matrix3x1<float> x(1.0f, 0.0f, 0.0f);
  cutlass::Quaternion<float> q = cutlass::Quaternion<float>::rotation(0, 0, 1, half_pi) * 2.0f;
  cutlass::Matrix3x1<float> v = q.rotate(x);

  float epsilon = 0.001f;

  EXPECT_TRUE(
    std::abs(v.at(0)) < epsilon &&
    std::abs(-v.at(1)) > (1 - epsilon) &&
    std::abs(v.at(2)) < epsilon
  );
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, spinor_rotation) {
  
  cutlass::Matrix3x1<float> x(1.0f, 0.0f, 0.0f);
  cutlass::Quaternion<float> q = cutlass::Quaternion<float>::rotation(0, 0, 1, half_pi);
  cutlass::Matrix3x1<float> v = cutlass::spinor_rotation(q, x);

  float epsilon = 0.001f;

  EXPECT_TRUE(
    std::abs(v.at(0)) < epsilon &&
    std::abs(v.at(1)) > (1 - epsilon) &&
    std::abs(v.at(2)) < epsilon
  );
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, spinor_rotation_inv) {
  
  cutlass::Matrix3x1<float> x(1.0f, 0.0f, 0.0f);
  cutlass::Quaternion<float> q = cutlass::Quaternion<float>::rotation(0, 0, 1, half_pi);
  cutlass::Matrix3x1<float> v = cutlass::spinor_rotation_inv(q, x);

  float epsilon = 0.001f;

  EXPECT_TRUE(
    std::abs(v.at(0)) < epsilon &&
    std::abs(-v.at(1)) > (1 - epsilon) &&
    std::abs(v.at(2)) < epsilon
  );
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, as_rotation_matrix3x3) {
  
  cutlass::Matrix3x1<float> x(1.0f, 0.0f, 0.0f);
  cutlass::Quaternion<float> q = cutlass::Quaternion<float>::rotation(0, 0, 1, half_pi);
  cutlass::Matrix3x1<float> v = q.as_rotation_matrix_3x3().product(x);

  float epsilon = 0.001f;

  EXPECT_TRUE(
    std::abs(v.at(0)) < epsilon &&
    std::abs(v.at(1)) > (1 - epsilon) &&
    std::abs(v.at(2)) < epsilon
  );
}


/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Quaternion, as_rotation_matrix4x4) {

  cutlass::Matrix4x1<float> x(1.0f, 0.0f, 0.0f, 1.0f);
  cutlass::Quaternion<float> q = cutlass::Quaternion<float>::rotation(0, 0, 1, half_pi);
  cutlass::Matrix4x1<float> v = q.as_rotation_matrix_4x4().product(x);

  float epsilon = 0.001f;

  EXPECT_TRUE(
    std::abs(v.at(0)) < epsilon &&
    std::abs(v.at(1)) > (1 - epsilon) &&
    std::abs(v.at(2)) < epsilon &&
    std::abs(v.at(3)) > (1 - epsilon)
  );
}

/////////////////////////////////////////////////////////////////////////////////////////////////

