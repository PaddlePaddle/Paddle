#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include <cstring>

#include "../common/cutlass_unit_test.h"

#include "cutlass/predicate_vector.h"
#include "cutlass/util/host_tensor.h"

namespace test {

template <typename PredicateVector>
__global__ void load_predicates(unsigned *output, unsigned const *input) {

  PredicateVector predicates;

  int const word_count = (PredicateVector::kPredicates + 31) / 32;

  int i = 0;
  for (int word_idx = 0; word_idx < word_count; ++word_idx) {
    unsigned word = input[word_idx];

    CUTLASS_PRAGMA_UNROLL
    for (int bit = 0; bit < sizeof(unsigned) * 8; ++bit) {
      bool pred = ((word >> bit) & 1);
      predicates.set(i, pred);

      if (predicates.at(i) != pred) {
        printf("ERROR - cannot read back predicate\n");
      }
      ++i;
    }
  }


  __syncthreads();

  i = 0;
  for (int word_idx = 0; word_idx < word_count; ++word_idx) {

    unsigned result = 0;
    for (int bit = 0; bit < sizeof(unsigned) * 8; ++bit) {
      bool pred = predicates.at(i ++);
      result |= (unsigned(pred) << bit);
    }
    output[word_idx] = result;
  }
}
}

TEST(PredicateVector, Basic) {

  static int const Bits = 32;
  static int const Words = (Bits + 31) / 32;

  typedef cutlass::PredicateVector<Bits> PredicateVector;

  cutlass::HostTensor<unsigned, cutlass::IdentityTensorLayout<1> > output;
  cutlass::HostTensor<unsigned, cutlass::IdentityTensorLayout<1>> input;

  output.reserve(Words);
  input.reserve(Words);

  // some arbitrary test bits
  unsigned values[] = {
    0xdeadbeef,
    0xa0070032,
    0x9076d001,
    0x00000000,
    0xabdfc0ad
  };

  for (int test = 0; test < 5; ++test) {

    input.host_data(0) = values[test];
    output.host_data(0) = 0;

    input.sync_device();
    output.sync_device();

    test::load_predicates<PredicateVector><<<
      dim3(1,1,1), dim3(1,1,1)
    >>>(
      output.device_data(),
      input.device_data()
    );

    output.sync_host();

    for (int word = 0; word < Words; ++word) {
      EXPECT_EQ(input.host_data(word), output.host_data(word))
        << "Expected: 0x" << std::hex << input.host_data(word)
        << ", got: 0x" << output.host_data(word)
        << std::dec;
    }
  }
}

TEST(PredicateVector, Count) {

  {
    typedef cutlass::PredicateVector<4, 8> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<4, 8> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<4, 4> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<4, 4> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<4, 2> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<4, 2> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<4, 1> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<4, 1> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<8, 8> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<8, 8> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<8, 4> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<8, 4> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<8, 2> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<8, 2> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<8, 1> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 2)
        << "PredicateVector<8, 1> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<16, 8> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<16, 8> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<16, 4> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<16, 4> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<16, 2> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 2)
        << "PredicateVector<16, 2> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<16, 1> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 4)
        << "PredicateVector<16, 1> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<32, 8> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 1)
        << "PredicateVector<32, 8> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<32, 4> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 2)
        << "PredicateVector<32, 4> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<32, 2> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 4)
        << "PredicateVector<32, 2> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<32, 1> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 8)
        << "PredicateVector<32, 1> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<64, 8> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 2)
        << "PredicateVector<64, 8> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<64, 4> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 4)
        << "PredicateVector<64, 4> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<64, 2> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 8)
        << "PredicateVector<64, 2> word count: " << int(PredicateVector::kWordCount);
  }

  {
    typedef cutlass::PredicateVector<64, 1> PredicateVector;
    EXPECT_EQ(int(PredicateVector::kWordCount), 16)
        << "PredicateVector<64, 1> word count: " << int(PredicateVector::kWordCount);
  }
}
