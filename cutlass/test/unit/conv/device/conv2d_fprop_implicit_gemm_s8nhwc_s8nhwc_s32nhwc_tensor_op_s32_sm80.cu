/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide Implicit GEMM interface
*/

#include "../../common/cutlass_unit_test.h"
#include "cutlass/cutlass.h"


#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "conv2d_testbed.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
////////////////////////////////////////////////////////////////////////////////
    
TEST(SM80_Device_Conv2d_Fprop_Analytic_ImplicitGemm_s8nhwc_s8nhwc_s32nhwc_tensor_op_s32,
  128x128_64x3_64x64x64) {

  /// Conv operation element types for the Gemm equivalent (ImplicitGemm)
  using ElementA           = int8_t;
  using ElementB           = int8_t;
  using ElementC           = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute     = float;

  using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
    ElementA, cutlass::layout::TensorNHWC,
    ElementB, cutlass::layout::TensorNHWC,
    ElementC, cutlass::layout::TensorNHWC,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 32>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      128 / cutlass::sizeof_bits<ElementC>::value,
      ElementAccumulator,
      ElementCompute
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAddSaturate,
    cutlass::conv::IteratorAlgorithm::kAnalytic
  >::Kernel;

  using Conv2dFprop = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

  /// Run all unit test sizes with device-level Conv2d instance
  EXPECT_TRUE(test::conv::device::TestAllConv2d<Conv2dFprop>());
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Conv2d_Fprop_Optimized_ImplicitGemm_s8nhwc_s8nhwc_s32nhwc_tensor_op_s32,
  128x128_64x3_64x64x64) {

  /// Conv operation element types for the Gemm equivalent (ImplicitGemm)
  using ElementA           = int8_t;
  using ElementB           = int8_t;
  using ElementC           = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute     = float;

  using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
    ElementA, cutlass::layout::TensorNHWC,
    ElementB, cutlass::layout::TensorNHWC,
    ElementC, cutlass::layout::TensorNHWC,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 32>,
    cutlass::epilogue::thread::LinearCombination<
      ElementC,
      128 / cutlass::sizeof_bits<ElementC>::value,
      ElementAccumulator,
      ElementCompute
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAddSaturate,
    cutlass::conv::IteratorAlgorithm::kOptimized
  >::Kernel;

  using Conv2dFprop = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

  /// Run all unit test sizes with device-level Conv2d instance
  EXPECT_TRUE(test::conv::device::TestAllConv2d<Conv2dFprop>());
}

////////////////////////////////////////////////////////////////////////////////
#endif  // CUTLASS_ARCH_MMA_SM80_SUPPORTED
