#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment

  
*/

#include <iostream>
#include <stdexcept>
#include <iomanip>
#include <ios>

#include "cutlass/core_io.h"

#include "cublas_helpers.h"
#include "rank_k_operation_profiler.h"
#include "gpu_timer.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {


/////////////////////////////////////////////////////////////////////////////////////////////////

/// Ctor
RankKOperationProfiler::RankKOperationProfiler(Options const &options): 
  OperationProfiler(
    options,
    library::OperationKind::kRankK,
    {
      {ArgumentTypeID::kEnumerated, {"rank_k_kind"}, "Variant of RankK (universal)"},
      {ArgumentTypeID::kInteger, {"n", "problem-size::n"}, "N dimension of the RankK problem space"},
      {ArgumentTypeID::kInteger, {"k", "problem-size::k"}, "K dimension of the RankK problem space"},
      {ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
      {ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
      {ArgumentTypeID::kEnumerated, {"fill_mode"}, "Fill Mode for RankK kernel (lower or upper)"},
      {ArgumentTypeID::kEnumerated, {"blas_mode"}, "Blas Mode for RankK kernel (symmetric or hermitian)"},
      {ArgumentTypeID::kScalar, {"alpha", "epilogue::alpha"}, "Epilogue scalar alpha"},
      {ArgumentTypeID::kScalar, {"beta", "epilogue::beta"}, "Epilogue scalar beta"},
      {ArgumentTypeID::kInteger, {"split_k_slices", "split-k-slices"}, "Number of partitions of K dimension"},
      {ArgumentTypeID::kInteger, {"batch_count", "batch-count"}, "Number of RankK computed in one batch"},
    },
    { library::Provider::kCUBLAS}
  ) {
  description_ = "      Rank-k Update. D = alpha * A*A^T + beta * C (symmetric) or D = alpha * A*A^H + beta * C (hermitian)";
}

/// Destructor
RankKOperationProfiler::~RankKOperationProfiler() {

}

/// Prints usage statement for the math function
void RankKOperationProfiler::print_usage(std::ostream &out) const {
  out << "RankK" << "\n\n";

  OperationProfiler::print_usage(out);
}

/// Prints examples
void RankKOperationProfiler::print_examples(std::ostream &out) const {

  out << "\nExamples:\n\n"
    << "Profile a particular problem size Syrk kernel:\n"
    << "  $ cutlass_profiler --operation=rank_k --blas_mode=symmetric --n=1024 --k=128\n\n"
    
    << "Profile a particular problem size Herk kernel:\n"
    << "  $ cutlass_profiler --operation=rank_k --blas_mode=hermitian --n=1024 --k=128\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=rank_k --n=1024:4096:256 --k=128:8192:128 --beta=0,1,2.5\n\n"

    << "Schmoo over accumulator types:\n"
    << "  $ cutlass_profiler --operation=rank_k --accumulator-type=f16,f32\n\n"

    << "Schmoo over fill modees:\n"
    << "  $ cutlass_profiler --operation=rank_k --fill_mode=lower/upper\n\n"

    << "Run when A is f16 with column-major or A is any datatype with row-major (For column major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=rank_k --A=f16:column or --A=*:row\n\n"

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=rank_k --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=rank_k --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=rank_k --dist=sequential,start:0,delta:1\n\n"

    << "Run a kernel with cta tile size of 256x128x32 and save workspace if results are incorrect (note that --cta-tile::k=32 is default cta-tile size):\n"
    << " $ cutlass_profiler --operation=rank_k --cta_m=256 --cta_n=128  --cta_k=32 --save-workspace=incorrect\n\n"
    
    << "Test your changes to rank_k kernels with a quick functional test and save results in functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=rank_k \\ \n"
    << "   --n=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --k=8,16,32,64,128,256,288,384,504,512,520 \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#if 0
// used this for debugging
static std::string byte_string(std::vector<uint8_t> const &bytes) {
  std::stringstream ss;

  ss << "0x";

  for (size_t idx = bytes.size(); idx > 0; --idx) {
    ss << std::hex << std::setw(2) << std::setfill('0') << uint32_t(bytes.at(idx - 1));
  }

  return ss.str();
}
#endif

Status RankKOperationProfiler::RankKProblem::parse(
  library::RankKDescription const &operation_desc,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {
  
  if (!arg_as_int(this->n, "n", problem_space, problem)) {
    // default value
    this->n = 1024;
  }
  
  if (!arg_as_int(this->k, "k", problem_space, problem)) {
    // default value
    this->k = 1024;
  }
  
  if (!arg_as_int(this->split_k_slices, "split_k_slices", problem_space, problem)) {
    // default value
    this->split_k_slices = 1;
  }
  
  if (!arg_as_int(this->batch_count, "batch_count", problem_space, problem)) {
    // default value
    this->batch_count = 1;
  }

  if (this->split_k_slices > 1 && this->batch_count > 1) {
    // At least one of these must be one
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
    this->alpha, 
    operation_desc.element_epilogue, 
    "alpha", 
    problem_space, 
    problem)) {

    if (!cast_from_double(this->alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }
  
  if (!arg_as_scalar(
    this->beta, 
    operation_desc.element_epilogue, 
    "beta", 
    problem_space, 
    problem)) {
    
    if (!cast_from_double(this->beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }
  
  this->lda = DeviceAllocation::get_packed_layout(
    operation_desc.A.layout, {int(this->n), int(this->k)}).front();

  this->ldc = DeviceAllocation::get_packed_layout(
    operation_desc.C.layout, {int(this->n), int(this->n)}).front();

  return Status::kSuccess;
}

/// Total number of bytes loaded
int64_t RankKOperationProfiler::RankKProblem::bytes(library::RankKDescription const &operation_desc) const {
  // Input bytes read and Output bytes written for the gemm problem
  int64_t bytes =
    int64_t(library::sizeof_bits(operation_desc.A.element) * n / 8) * k +
    int64_t(library::sizeof_bits(operation_desc.A.element) * n / 8) * k +
    // Half matrix including the diagonal will have (N*(N+1))/2 elements
    int64_t(library::sizeof_bits(operation_desc.C.element) * n / 8) * (n+1) / 2;

  // Set is_beta_zero true if beta is zero
  bool is_beta_zero = std::all_of(beta.begin(), beta.end(), [](uint8_t i) { return i==0; });

  // Output bytes read for the gemm problem for non-zero beta values
  if (!is_beta_zero) {
    bytes += int64_t(library::sizeof_bits(operation_desc.C.element) * n / 8) * (n+1) / 2;
  }

  bytes *= batch_count;

  return bytes;
}

/// Total number of flops computed
int64_t RankKOperationProfiler::RankKProblem::flops(library::RankKDescription const &operation_desc) const {

  // FLOPs = 2 * n(n+1)k/2 [mma] + 2 * n(n+1)/2 [epilogue]
  // FLOPs = n(n+1)(k + 1)
  int64_t flops_ = n * (n + 1) * (k + 1);

  // complex-valued support
  switch (operation_desc.tile_description.math_instruction.math_operation) {
  case library::MathOperationID::kMultiplyAddComplex:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddComplexFastF32:
    flops_ *= 4;
    break;
    
  case library::MathOperationID::kMultiplyAddGaussianComplex:
    flops_ *= 3;
    break;

  default: break;
  }

  return flops_;
}

/// Initializes a performance result
void RankKOperationProfiler::RankKProblem::initialize_result(
  PerformanceResult &result,
  library::RankKDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.arguments.resize(problem_space.rank());

  set_argument(result, "rank_k_kind", problem_space, library::to_string(operation_desc.rank_k_kind));

  set_argument(result, "A", problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" + library::to_string(operation_desc.A.layout));

  set_argument(result, "C", problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" + library::to_string(operation_desc.C.layout));

  set_argument(result, "fill_mode", problem_space, library::to_string(operation_desc.fill_mode));

  set_argument(result, "blas_mode", problem_space, library::to_string(operation_desc.blas_mode));

  set_argument(result, "n", problem_space, n);
  set_argument(result, "k", problem_space, k);

  set_argument(result, "split_k_slices", problem_space, split_k_slices);
  set_argument(result, "batch_count", problem_space, batch_count);

  set_argument(result, "alpha", problem_space,
    library::lexical_cast(alpha, operation_desc.element_epilogue));

  set_argument(result, "beta", problem_space,
    library::lexical_cast(beta, operation_desc.element_epilogue));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Extracts the problem dimensions
Status RankKOperationProfiler::initialize_configuration(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::RankKDescription const &operation_desc = 
    static_cast<library::RankKDescription const &>(operation->description());

  if (operation_desc.rank_k_kind != library::RankKKind::kUniversal) {
    return Status::kErrorInvalidProblem;
  }

  Status status = problem_.parse(operation_desc, problem_space, problem);
  
  if (status != Status::kSuccess) {
    return status;
  }

  rank_k_workspace_.configuration.problem_size.m() = int(problem_.n);
  rank_k_workspace_.configuration.problem_size.n() = int(problem_.n);
  rank_k_workspace_.configuration.problem_size.k() = int(problem_.k);
  rank_k_workspace_.configuration.lda = problem_.lda;
  rank_k_workspace_.configuration.ldc = problem_.ldc;
  rank_k_workspace_.configuration.ldd = problem_.ldc;
  //rank_k_workspace_.configuration.split_k_slices = int(problem_.split_k_slices);
  rank_k_workspace_.configuration.batch_count = int(problem_.split_k_slices);

  rank_k_workspace_.arguments.A = nullptr;
  rank_k_workspace_.arguments.C = nullptr;
  rank_k_workspace_.arguments.D = nullptr;
  rank_k_workspace_.arguments.alpha = problem_.alpha.data();
  rank_k_workspace_.arguments.beta = problem_.beta.data();
  rank_k_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  initialize_result_(this->model_result_, options, operation_desc, problem_space);
  
  return operation->can_implement(&rank_k_workspace_.configuration, &rank_k_workspace_.arguments);
}

/// Initializes the performance result
void RankKOperationProfiler::initialize_result_(
  PerformanceResult &result,
  Options const &options,  
  library::RankKDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.provider = library::Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;
  
  problem_.initialize_result(result, operation_desc, problem_space);

  OperationProfiler::initialize_result_(result, operation_desc, problem_space);


  result.bytes = problem_.bytes(operation_desc);
  result.flops = problem_.flops(operation_desc);

  result.runtime = 0;

  // complex-valued support
  switch (operation_desc.tile_description.math_instruction.math_operation) {
  case library::MathOperationID::kMultiplyAddComplex:
    result.flops *= 4;
    break;
     
  case library::MathOperationID::kMultiplyAddComplexFastF32:
    result.flops *= 4;
    break;

  default: break;
  }

}

/// Initializes workspace
Status RankKOperationProfiler::initialize_workspace(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {
  
  library::RankKDescription const &operation_desc = 
    static_cast<library::RankKDescription const &>(operation->description());

  if (options.execution_mode != ExecutionMode::kDryRun) {

    rank_k_workspace_.A = device_context.allocate_tensor(
      options,
      "A",
      operation_desc.A.element,
      operation_desc.A.layout,
      {int(problem_.n), int(problem_.k)},
      {int(problem_.lda)}
    );

    rank_k_workspace_.C = device_context.allocate_tensor(
      options,
      "C",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.n), int(problem_.n)},
      {int(problem_.ldc)},
      1 // batch_count = 1, default
    );

    rank_k_workspace_.Computed = device_context.allocate_tensor(
      "D",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.n), int(problem_.n)},
      {int(problem_.ldc)}
    );

    rank_k_workspace_.Reference = device_context.allocate_tensor(
      "Reference",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.n), int(problem_.n)},
      {int(problem_.ldc)}
    );

    rank_k_workspace_.Computed->copy_from_device(rank_k_workspace_.C->data());
    rank_k_workspace_.Reference->copy_from_device(rank_k_workspace_.C->data());
  }


  //
  // Initialize the CUTLASS operation
  //
  Status status = Status::kSuccess;

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    if (options.execution_mode != ExecutionMode::kDryRun) {

      uint64_t workspace_size = operation->get_host_workspace_size(&rank_k_workspace_.configuration);
      rank_k_workspace_.host_workspace.resize(workspace_size, 0);

      workspace_size = operation->get_device_workspace_size(&rank_k_workspace_.configuration);
      rank_k_workspace_.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

      status = operation->initialize(
        &rank_k_workspace_.configuration,
        rank_k_workspace_.host_workspace.data(),
        rank_k_workspace_.device_workspace.data());
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //
    results_.push_back(model_result_);
    results_.back().provider = library::Provider::kCUTLASS;
    results_.back().op_kind = library::OperationKind::kRankK;
    results_.back().disposition = Disposition::kNotRun;

    for(auto provider : verification_providers_) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
    }
  }

  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool RankKOperationProfiler::verify_cutlass(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (!options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  // Initialize structure containing RankK arguments
  rank_k_workspace_.arguments.A = rank_k_workspace_.A->data();
  rank_k_workspace_.arguments.C = rank_k_workspace_.C->data();
  rank_k_workspace_.arguments.D = rank_k_workspace_.Computed->data();
  rank_k_workspace_.arguments.alpha = problem_.alpha.data();
  rank_k_workspace_.arguments.beta = problem_.beta.data();
  rank_k_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  //
  // Run the CUTLASS operation
  //

  results_.back().status = operation->run(
    &rank_k_workspace_.arguments, 
    rank_k_workspace_.host_workspace.data(),
    rank_k_workspace_.device_workspace.data());

  if (results_.back().status != Status::kSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // CUTLASS op ran the but not yet verified against any verification provider
  results_.back().disposition = Disposition::kNotVerified;

  //
  // Run verification providers
  //

  if (options.verification.enabled) {

#if CUTLASS_ENABLE_CUBLAS
    if (options.verification.provider_enabled(library::Provider::kCUBLAS)) {

      // Guard against unsupported cases
      auto const & rank_k_desc = static_cast<library::RankKDescription const &>(operation->description());

      if (cublas_satisfies(rank_k_desc) == Status::kSuccess) {

        // call cublas verification if supported
        verify_with_cublas_(
          options,
          report,
          device_context,
          operation,
          problem_space,
          problem);
        }

      else {
        // set verification map for cublas to not supported
        results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotSupported;
      }
    }
#endif // #if CUTLASS_ENABLE_CUBLAS
    
    // Update disposition to worst case verification outcome among all 
    // verification providers which are supported
    bool is_any_verification_run_passed = false;
    for(auto &m : results_.back().verification_map) {
      if(m.second == Disposition::kFailed || m.second == Disposition::kIncorrect) {
        results_.back().disposition = m.second;
        return true;
      }
      if(!is_any_verification_run_passed && m.second == Disposition::kPassed) {
        is_any_verification_run_passed = true;
      }
    }

    if(is_any_verification_run_passed) {
      results_.back().disposition = Disposition::kPassed;
    }
  }

  // Return true means continue profiling
  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool RankKOperationProfiler::verify_with_cublas_(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {


#if CUTLASS_ENABLE_CUBLAS

  library::RankKDescription const &rank_k_desc = 
    static_cast<library::RankKDescription const &>(operation->description());

  //
  // Construct cuBLAS operators
  //
    
  CublasCreate handle;
  hipblasStatus_t status = handle.get_cublas_create_status();

  if (status != HIPBLAS_STATUS_SUCCESS) {

    results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kFailed;
    return true;
  }

  //
  // Initialize state
  //

  try {

    //
    // Construct dispatcher to cublas<t>Syrk()
    //

    // Initialize structure containing RankK arguments
    rank_k_workspace_.arguments.A = rank_k_workspace_.A->data();
    rank_k_workspace_.arguments.C = rank_k_workspace_.Reference->data();
    rank_k_workspace_.arguments.D = rank_k_workspace_.Reference->data();
    rank_k_workspace_.arguments.alpha = problem_.alpha.data();
    rank_k_workspace_.arguments.beta = problem_.beta.data();
    rank_k_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    detail::cublasRankKDispatcher rank_k_op( 
      rank_k_desc, 
      rank_k_workspace_.configuration,
      rank_k_workspace_.arguments
    );

    if (rank_k_op.status != Status::kSuccess) {
      results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotRun;
      return true;
    }

    results_.back().status = Status::kSuccess;

    status = rank_k_op(handle);

    // Handle errors
    if (status != HIPBLAS_STATUS_SUCCESS) {

      results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kFailed;
      return true;
    }

    //
    // Verify results
    //

    results_.back().verification_map[library::Provider::kCUBLAS] = compare_tensors(
      options,
      *rank_k_workspace_.Computed,
      *rank_k_workspace_.Reference
    );

    // Save workspace if incorrect
    if (options.verification.save_workspace == SaveWorkspace::kIncorrect && 
      results_.back().verification_map[library::Provider::kCUBLAS] == Disposition::kIncorrect) {

      save_workspace(
        device_context,
        options,
        rank_k_desc,
        library::Provider::kCUTLASS,
        library::Provider::kCUBLAS);
    }
  }
  catch (...) {
    results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kFailed;
  }

#endif

  // Return true means continue profiling
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool RankKOperationProfiler::profile(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    // Initialize structure containing RankK arguments
    rank_k_workspace_.arguments.A = rank_k_workspace_.A->data();
    rank_k_workspace_.arguments.C = rank_k_workspace_.C->data();
    rank_k_workspace_.arguments.D = rank_k_workspace_.Computed->data();
    rank_k_workspace_.arguments.alpha = problem_.alpha.data();
    rank_k_workspace_.arguments.beta = problem_.beta.data();
    rank_k_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    results_.back().status = profile_cutlass_(
      results_.back().runtime,
      options,
      operation,
      &rank_k_workspace_.arguments,
      rank_k_workspace_.host_workspace.data(),
      rank_k_workspace_.device_workspace.data()
    );
  }
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
