/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <cstring>

#include "cutlass/numeric_types.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/layout/tensor.h"

#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"

#include "cutlass/library/util.h"

#include "device_allocation.h"

namespace cutlass {
namespace profiler {

/////////////////////////////////////////////////////////////////////////////////////////////////

size_t DeviceAllocation::bytes(library::NumericTypeID type, size_t capacity) {
  return size_t(cutlass::library::sizeof_bits(type)) * capacity / 8;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout>
static std::vector<int64_t> get_packed_layout_stride(std::vector<int> const &extent) {

  typename Layout::TensorCoord extent_coord;
  typename Layout::Stride stride_coord;

  if (extent.size() != size_t(Layout::kRank)) {
    throw std::runtime_error("Layout does not have same rank as extent vector.");
  }

  for (int i = 0; i < Layout::kRank; ++i) {
    extent_coord[i] = extent.at(i);
  }

  std::vector<int64_t> stride;
  stride.resize(Layout::kStrideRank, 0);

  Layout layout = Layout::packed(extent_coord);
  stride_coord = layout.stride();

  for (int i = 0; i < Layout::kStrideRank; ++i) {
    stride.at(i) = (int64_t)stride_coord[i];
  }

  return stride;
}

/// Returns the stride of a packed layout
std::vector<int64_t> DeviceAllocation::get_packed_layout(
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent) {

  std::vector<int64_t> stride;

  switch (layout_id) {
    case library::LayoutTypeID::kColumnMajor: 
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajor>(extent);
      break;
    case library::LayoutTypeID::kRowMajor: 
      stride = get_packed_layout_stride<cutlass::layout::RowMajor>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK2:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<2>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK2:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<2>>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK4:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<4>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK4:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<4>>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK16:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<16>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK16:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<16>>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK32:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<32>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK32:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<32>>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK64:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<64>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK64:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<64>>(extent);
      break;
    case library::LayoutTypeID::kTensorNCHW:
      stride = get_packed_layout_stride<cutlass::layout::TensorNCHW>(extent);
      break;
    case library::LayoutTypeID::kTensorNHWC:
      stride = get_packed_layout_stride<cutlass::layout::TensorNHWC>(extent);
      break;
    case library::LayoutTypeID::kTensorNDHWC:
      stride = get_packed_layout_stride<cutlass::layout::TensorNDHWC>(extent);
      break;
    case library::LayoutTypeID::kTensorNC32HW32:
      stride = get_packed_layout_stride<cutlass::layout::TensorNCxHWx<32>>(extent);
      break;
    case library::LayoutTypeID::kTensorNC64HW64:
      stride = get_packed_layout_stride<cutlass::layout::TensorNCxHWx<64>>(extent);
      break;
    case library::LayoutTypeID::kTensorC32RSK32:
      stride = get_packed_layout_stride<cutlass::layout::TensorCxRSKx<32>>(extent);
      break;
    case library::LayoutTypeID::kTensorC64RSK64:
      stride = get_packed_layout_stride<cutlass::layout::TensorCxRSKx<64>>(extent);
      break;
    default: break;
  }

  return stride;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Template to use CUTLASS Layout functions to 
template <typename Layout>
static size_t construct_layout_(
  void *bytes,
  library::LayoutTypeID layout_id,
  std::vector<int> const &extent,
  std::vector<int64_t> &stride) {

  if (extent.size() != Layout::kRank) {
    throw std::runtime_error(
      "Layout must have same rank as extent vector.");
  }

  if (Layout::kStrideRank && stride.empty()) {

    stride = get_packed_layout_stride<Layout>(extent);

    return construct_layout_<Layout>(
      bytes, 
      layout_id, 
      extent,
      stride);
  }
  else if (Layout::kStrideRank && stride.size() != Layout::kStrideRank) {
    throw std::runtime_error(
      "Layout requires either empty stride or stride vector matching Layout::kStrideRank");
  }

  typename Layout::Stride stride_coord;
  for (int i = 0; i < Layout::kStrideRank; ++i) {
    stride_coord[i] = (int)stride.at(i);
  }

  typename Layout::TensorCoord extent_coord;
  for (int i = 0; i < Layout::kRank; ++i) {
    extent_coord[i] = extent.at(i);
  }

  // Construct the CUTLASS layout object from the stride object
  Layout layout(stride_coord);

  // Pack it into bytes
  if (bytes) {
    *reinterpret_cast<Layout *>(bytes) = layout; 
  }

  // Return capacity
  size_t capacity_ = layout.capacity(extent_coord);

  return capacity_;
}

/// returns the capacity needed
size_t DeviceAllocation::construct_layout(
  void *bytes,
  library::LayoutTypeID layout_id,
  std::vector<int> const &extent,
  std::vector<int64_t> &stride) {

  switch (layout_id) {
    case library::LayoutTypeID::kColumnMajor: 
      return construct_layout_<cutlass::layout::ColumnMajor>(bytes, layout_id, extent, stride);
      
    case library::LayoutTypeID::kRowMajor: 
      return construct_layout_<cutlass::layout::RowMajor>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK2:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<2>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK2:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<2>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK4:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<4>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK4:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<4>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK16:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<16>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK16:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<16>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK32:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<32>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK32:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<32>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK64:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<64>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK64:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<64>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNCHW:
      return construct_layout_<cutlass::layout::TensorNHWC>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNHWC:
      return construct_layout_<cutlass::layout::TensorNHWC>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNDHWC:
      return construct_layout_<cutlass::layout::TensorNDHWC>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNC32HW32:
      return construct_layout_<cutlass::layout::TensorNCxHWx<32>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNC64HW64:
      return construct_layout_<cutlass::layout::TensorNCxHWx<64>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorC32RSK32:
      return construct_layout_<cutlass::layout::TensorCxRSKx<32>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorC64RSK64:
      return construct_layout_<cutlass::layout::TensorCxRSKx<64>>(bytes, layout_id, extent, stride);

    default: break;
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

DeviceAllocation::DeviceAllocation(): 
  type_(library::NumericTypeID::kInvalid), 
  batch_stride_(0),
  capacity_(0), 
  pointer_(nullptr),
  layout_(library::LayoutTypeID::kUnknown),
  batch_count_(1) {

}

DeviceAllocation::DeviceAllocation(
  library::NumericTypeID type, 
  size_t capacity
):
  type_(type), batch_stride_(capacity), capacity_(capacity), pointer_(nullptr), 
  layout_(library::LayoutTypeID::kUnknown), batch_count_(1) {

  hipError_t result = hipMalloc((void **)&pointer_, bytes(type, capacity));

  if (result != hipSuccess) {
    type_ = library::NumericTypeID::kInvalid;
    capacity_ = 0;
    pointer_ = nullptr;
    throw std::bad_alloc();
  }
}

DeviceAllocation::DeviceAllocation(
  library::NumericTypeID type, 
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent, 
  std::vector<int64_t> const &stride,
  int batch_count
):
  type_(type), batch_stride_(size_t(0)), capacity_(size_t(0)), pointer_(nullptr), batch_count_(1) {

  reset(type, layout_id, extent, stride, batch_count);
}

DeviceAllocation::~DeviceAllocation() {
  if (pointer_) {
    hipFree(pointer_);
  }
}

DeviceAllocation &DeviceAllocation::reset() {
  if (pointer_) {
    hipFree(pointer_);
  }

  type_ = library::NumericTypeID::kInvalid;
  batch_stride_ = 0;
  capacity_ = 0;
  pointer_ = nullptr;
  layout_ = library::LayoutTypeID::kUnknown;
  stride_.clear();
  extent_.clear();
  tensor_ref_buffer_.clear();
  batch_count_ = 1;

  return *this;
}

DeviceAllocation &DeviceAllocation::reset(library::NumericTypeID type, size_t capacity) {

  reset();

  type_ = type;
  batch_stride_ = capacity;
  capacity_ = capacity;

  hipError_t result = hipMalloc((void **)&pointer_, bytes(type_, capacity_));
  if (result != hipSuccess) {
    throw std::bad_alloc();
  }

  layout_ = library::LayoutTypeID::kUnknown;
  stride_.clear();
  extent_.clear();
  batch_count_ = 1;

  tensor_ref_buffer_.resize(sizeof(pointer_), 0);
  std::memcpy(tensor_ref_buffer_.data(), &pointer_, sizeof(pointer_));

  return *this;
}

/// Allocates memory for a given layout and tensor
DeviceAllocation &DeviceAllocation::reset(
  library::NumericTypeID type, 
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent, 
  std::vector<int64_t> const &stride,
  int batch_count) {

  reset();

  tensor_ref_buffer_.resize(sizeof(pointer_) + (sizeof(int64_t) * library::get_layout_stride_rank(layout_id)), 0);

  type_ = type;

  layout_ = layout_id;
  stride_ = stride;
  extent_ = extent;
  batch_count_ = batch_count;

  batch_stride_ = construct_layout(
    tensor_ref_buffer_.data() + sizeof(pointer_), 
    layout_id, 
    extent, 
    stride_);

  capacity_ = batch_stride_ * batch_count_;

  hipError_t result = hipMalloc((void **)&pointer_, bytes(type, capacity_));
  if (result != hipSuccess) {
    throw std::bad_alloc();
  }

  std::memcpy(tensor_ref_buffer_.data(), &pointer_, sizeof(pointer_));

  return *this;
}

bool DeviceAllocation::good() const {
  return (capacity_ && pointer_);
}

library::NumericTypeID DeviceAllocation::type() const {
  return type_;
}

void *DeviceAllocation::data() const {
  return pointer_;
}

void *DeviceAllocation::batch_data(int batch_idx) const {
    return static_cast<char *>(data()) + batch_stride_bytes() * batch_idx; 
}

library::LayoutTypeID DeviceAllocation::layout() const {
  return layout_;
}

std::vector<int64_t> const & DeviceAllocation::stride() const {
  return stride_;
}

/// Gets the extent vector
std::vector<int> const & DeviceAllocation::extent() const {
  return extent_;
}

/// Gets the number of adjacent tensors in memory
int DeviceAllocation::batch_count() const {
  return batch_count_;
}

/// Gets the stride (in units of elements) beteween items
int64_t DeviceAllocation::batch_stride() const {
  return batch_stride_;
}

/// Gets the stride (in units of bytes) beteween items
int64_t DeviceAllocation::batch_stride_bytes() const {
  return bytes(type_, batch_stride_);
}

size_t DeviceAllocation::capacity() const {
  return capacity_;
}

size_t DeviceAllocation::bytes() const {
  return bytes(type_, capacity_);
}

/// Copies from an equivalent-sized tensor in device memory
void DeviceAllocation::copy_from_device(void const *ptr) {
  hipError_t result = hipMemcpy(data(), ptr, bytes(), hipMemcpyDeviceToDevice);
  if (result != hipSuccess) {
    throw std::runtime_error("Failed device-to-device copy");
  }
}

/// Copies from an equivalent-sized tensor in device memory
void DeviceAllocation::copy_from_host(void const *ptr) {
  hipError_t result = hipMemcpy(data(), ptr, bytes(), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    throw std::runtime_error("Failed device-to-device copy");
  }
}

/// Copies from an equivalent-sized tensor in device memory
void DeviceAllocation::copy_to_host(void *ptr) {
  hipError_t result = hipMemcpy(ptr, data(), bytes(), hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    throw std::runtime_error("Failed device-to-device copy");
  }
}

void DeviceAllocation::initialize_random_device(int seed, Distribution dist) {
  if (!good()) {
    throw std::runtime_error("Attempting to initialize invalid allocation.");
  }

  // Instantiate calls to CURAND here. This file takes a long time to compile for
  // this reason.

  switch (type_) {
  case library::NumericTypeID::kF16:
    cutlass::reference::device::BlockFillRandom<cutlass::half_t>(
      reinterpret_cast<cutlass::half_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kBF16:
    cutlass::reference::device::BlockFillRandom<cutlass::bfloat16_t>(
      reinterpret_cast<cutlass::bfloat16_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kTF32:
    cutlass::reference::device::BlockFillRandom<cutlass::tfloat32_t>(
      reinterpret_cast<cutlass::tfloat32_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF32:
    cutlass::reference::device::BlockFillRandom<float>(
      reinterpret_cast<float *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCBF16:
    cutlass::reference::device::BlockFillRandom<complex<bfloat16_t>>(
      reinterpret_cast<complex<bfloat16_t> *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCTF32:
    cutlass::reference::device::BlockFillRandom<cutlass::complex<cutlass::tfloat32_t>>(
      reinterpret_cast<cutlass::complex<cutlass::tfloat32_t> *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCF32:
    cutlass::reference::device::BlockFillRandom<cutlass::complex<float>>(
      reinterpret_cast<cutlass::complex<float> *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF64:
    cutlass::reference::device::BlockFillRandom<double>(
      reinterpret_cast<double *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCF64:
    cutlass::reference::device::BlockFillRandom<complex<double>>(
      reinterpret_cast<complex<double> *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS2:
    cutlass::reference::device::BlockFillRandom<int2b_t>(
      reinterpret_cast<int2b_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS4:
    cutlass::reference::device::BlockFillRandom<int4b_t>(
      reinterpret_cast<int4b_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS8:
    cutlass::reference::device::BlockFillRandom<int8_t>(
      reinterpret_cast<int8_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS16:
    cutlass::reference::device::BlockFillRandom<int16_t>(
      reinterpret_cast<int16_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS32:
    cutlass::reference::device::BlockFillRandom<int32_t>(
      reinterpret_cast<int32_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS64:
    cutlass::reference::device::BlockFillRandom<int64_t>(
      reinterpret_cast<int64_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kB1:
    cutlass::reference::device::BlockFillRandom<uint1b_t>(
      reinterpret_cast<uint1b_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU2:
    cutlass::reference::device::BlockFillRandom<uint2b_t>(
      reinterpret_cast<uint2b_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU4:
    cutlass::reference::device::BlockFillRandom<uint4b_t>(
      reinterpret_cast<uint4b_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU8:
    cutlass::reference::device::BlockFillRandom<uint8_t>(
      reinterpret_cast<uint8_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU16:
    cutlass::reference::device::BlockFillRandom<uint16_t>(
      reinterpret_cast<uint16_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU32:
    cutlass::reference::device::BlockFillRandom<uint32_t>(
      reinterpret_cast<uint32_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU64:
    cutlass::reference::device::BlockFillRandom<uint64_t>(
      reinterpret_cast<uint64_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  default: break;
  }
}

void DeviceAllocation::initialize_random_host(int seed, Distribution dist) {
  if (!good()) {
    throw std::runtime_error("Attempting to initialize invalid allocation.");
  }

  std::vector<uint8_t> host_data(bytes());

  switch (type_) {
  case library::NumericTypeID::kF16:
    cutlass::reference::host::BlockFillRandom<cutlass::half_t>(
      reinterpret_cast<cutlass::half_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kBF16:
    cutlass::reference::host::BlockFillRandom<cutlass::bfloat16_t>(
      reinterpret_cast<cutlass::bfloat16_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kTF32:
    cutlass::reference::host::BlockFillRandom<cutlass::tfloat32_t>(
      reinterpret_cast<cutlass::tfloat32_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF32:
    cutlass::reference::host::BlockFillRandom<float>(
      reinterpret_cast<float *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCF16:
    cutlass::reference::host::BlockFillRandom<cutlass::complex<cutlass::half_t>>(
      reinterpret_cast<cutlass::complex<cutlass::half_t> *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCBF16:
    cutlass::reference::host::BlockFillRandom<cutlass::complex<cutlass::bfloat16_t>>(
      reinterpret_cast<cutlass::complex<cutlass::bfloat16_t> *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCTF32:
    cutlass::reference::host::BlockFillRandom<cutlass::complex<cutlass::tfloat32_t>>(
      reinterpret_cast<cutlass::complex<cutlass::tfloat32_t> *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCF32:
    cutlass::reference::host::BlockFillRandom<cutlass::complex<float>>(
      reinterpret_cast<cutlass::complex<float> *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF64:
    cutlass::reference::host::BlockFillRandom<double>(
      reinterpret_cast<double *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kCF64:
    cutlass::reference::host::BlockFillRandom<cutlass::complex<double>>(
      reinterpret_cast<cutlass::complex<double> *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS2:
    cutlass::reference::host::BlockFillRandom<int2b_t>(
      reinterpret_cast<int2b_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS4:
    cutlass::reference::host::BlockFillRandom<int4b_t>(
      reinterpret_cast<int4b_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS8:
    cutlass::reference::host::BlockFillRandom<int8_t>(
      reinterpret_cast<int8_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS16:
    cutlass::reference::host::BlockFillRandom<int16_t>(
      reinterpret_cast<int16_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS32:
    cutlass::reference::host::BlockFillRandom<int32_t>(
      reinterpret_cast<int32_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS64:
    cutlass::reference::host::BlockFillRandom<int64_t>(
      reinterpret_cast<int64_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kB1:
    cutlass::reference::host::BlockFillRandom<uint1b_t>(
      reinterpret_cast<uint1b_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU2:
    cutlass::reference::host::BlockFillRandom<uint2b_t>(
      reinterpret_cast<uint2b_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU4:
    cutlass::reference::host::BlockFillRandom<uint4b_t>(
      reinterpret_cast<uint4b_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU8:
    cutlass::reference::host::BlockFillRandom<uint8_t>(
      reinterpret_cast<uint8_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU16:
    cutlass::reference::host::BlockFillRandom<uint16_t>(
      reinterpret_cast<uint16_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU32:
    cutlass::reference::host::BlockFillRandom<uint32_t>(
      reinterpret_cast<uint32_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU64:
    cutlass::reference::host::BlockFillRandom<uint64_t>(
      reinterpret_cast<uint64_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  default: break;
  }

  copy_from_host(host_data.data());
}

void DeviceAllocation::initialize_random_sparsemeta_device(int seed, int MetaSizeInBits) {
  if (!good()) {
    throw std::runtime_error("Attempting to initialize invalid allocation.");
  }

  // Instantiate calls to CURAND here. This file takes a long time to compile for
  // this reason.

  switch (type_) {
  case library::NumericTypeID::kU16:
    cutlass::reference::device::BlockFillRandomSparseMeta<uint16_t>(
      reinterpret_cast<uint16_t *>(pointer_),
      capacity_,
      seed,
      MetaSizeInBits
    );
    break;
  case library::NumericTypeID::kU32:
    cutlass::reference::device::BlockFillRandomSparseMeta<uint32_t>(
      reinterpret_cast<uint32_t *>(pointer_),
      capacity_,
      seed,
      MetaSizeInBits
    );
    break;
  default:
    break;
  }
}

void DeviceAllocation::initialize_random_sparsemeta_host(int seed, int MetaSizeInBits) {
  if (!good()) {
    throw std::runtime_error("Attempting to initialize invalid allocation.");
  }

  std::vector<uint8_t> host_data(bytes());

  switch (type_) {
  case library::NumericTypeID::kS16:
    cutlass::reference::host::BlockFillRandomSparseMeta<uint16_t>(
      reinterpret_cast<uint16_t *>(host_data.data()),
      capacity_,
      seed,
      MetaSizeInBits
    );
    break;
  case library::NumericTypeID::kS32:
    cutlass::reference::host::BlockFillRandomSparseMeta<uint32_t>(
      reinterpret_cast<uint32_t *>(host_data.data()),
      capacity_,
      seed,
      MetaSizeInBits
    );
    break;
  default:
    break;
  }

  copy_from_host(host_data.data());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns true if two blocks have exactly the same value
bool DeviceAllocation::block_compare_equal(
  library::NumericTypeID numeric_type, 
  void const *ptr_A, 
  void const *ptr_B, 
  size_t capacity) {

  switch (numeric_type) {
  case library::NumericTypeID::kF16:
    return reference::device::BlockCompareEqual<half_t>(
      reinterpret_cast<half_t const *>(ptr_A), 
      reinterpret_cast<half_t const *>(ptr_B), 
      capacity);
    
  case library::NumericTypeID::kBF16:
    return reference::device::BlockCompareEqual<bfloat16_t>(
      reinterpret_cast<bfloat16_t const *>(ptr_A), 
      reinterpret_cast<bfloat16_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kTF32:
    return reference::device::BlockCompareEqual<tfloat32_t>(
      reinterpret_cast<tfloat32_t const *>(ptr_A), 
      reinterpret_cast<tfloat32_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kF32:
    return reference::device::BlockCompareEqual<float>(
      reinterpret_cast<float const *>(ptr_A), 
      reinterpret_cast<float const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kCF32:
    return reference::device::BlockCompareEqual<cutlass::complex<float> >(
      reinterpret_cast<complex<float> const *>(ptr_A), 
      reinterpret_cast<complex<float> const *>(ptr_B), 
      capacity);
  
  case library::NumericTypeID::kCF16:
    return reference::device::BlockCompareEqual<complex<half_t>>(
      reinterpret_cast<complex<half_t> const *>(ptr_A), 
      reinterpret_cast<complex<half_t> const *>(ptr_B), 
      capacity);
    
  case library::NumericTypeID::kCBF16:
    return reference::device::BlockCompareEqual<complex<bfloat16_t>>(
      reinterpret_cast<complex<bfloat16_t> const *>(ptr_A), 
      reinterpret_cast<complex<bfloat16_t> const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kCTF32:
    return reference::device::BlockCompareEqual<complex<tfloat32_t>>(
      reinterpret_cast<complex<tfloat32_t> const *>(ptr_A), 
      reinterpret_cast<complex<tfloat32_t> const *>(ptr_B), 
      capacity);
  
  case library::NumericTypeID::kF64:
    return reference::device::BlockCompareEqual<double>(
      reinterpret_cast<double const *>(ptr_A), 
      reinterpret_cast<double const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kCF64:
    return reference::device::BlockCompareEqual<complex<double>>(
      reinterpret_cast<complex<double> const *>(ptr_A), 
      reinterpret_cast<complex<double> const *>(ptr_B), 
      capacity);
  
  case library::NumericTypeID::kS2:
    return reference::device::BlockCompareEqual<int2b_t>(
      reinterpret_cast<int2b_t const *>(ptr_A), 
      reinterpret_cast<int2b_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS4:
    return reference::device::BlockCompareEqual<int4b_t>(
      reinterpret_cast<int4b_t const *>(ptr_A), 
      reinterpret_cast<int4b_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS8:
    return reference::device::BlockCompareEqual<int8_t>(
      reinterpret_cast<int8_t const *>(ptr_A), 
      reinterpret_cast<int8_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS16:
    return reference::device::BlockCompareEqual<int16_t>(
      reinterpret_cast<int16_t const *>(ptr_A), 
      reinterpret_cast<int16_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS32:
    return reference::device::BlockCompareEqual<int32_t>(
      reinterpret_cast<int32_t const *>(ptr_A), 
      reinterpret_cast<int32_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS64:
    return reference::device::BlockCompareEqual<int64_t>(
      reinterpret_cast<int64_t const *>(ptr_A), 
      reinterpret_cast<int64_t const *>(ptr_B), 
      capacity);
  
  case library::NumericTypeID::kB1:
    return reference::device::BlockCompareEqual<uint1b_t>(
      reinterpret_cast<uint1b_t const *>(ptr_A), 
      reinterpret_cast<uint1b_t const *>(ptr_B), 
      capacity);
  
  case library::NumericTypeID::kU2:
    return reference::device::BlockCompareEqual<uint2b_t>(
      reinterpret_cast<uint2b_t const *>(ptr_A), 
      reinterpret_cast<uint2b_t const *>(ptr_B), 
      capacity);
  
  case library::NumericTypeID::kU4:
    return reference::device::BlockCompareEqual<uint4b_t>(
      reinterpret_cast<uint4b_t const *>(ptr_A), 
      reinterpret_cast<uint4b_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU8:
    return reference::device::BlockCompareEqual<uint8_t>(
      reinterpret_cast<uint8_t const *>(ptr_A), 
      reinterpret_cast<uint8_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU16:
    return reference::device::BlockCompareEqual<uint16_t>(
      reinterpret_cast<uint16_t const *>(ptr_A), 
      reinterpret_cast<uint16_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU32:
    return reference::device::BlockCompareEqual<uint32_t>(
      reinterpret_cast<uint32_t const *>(ptr_A), 
      reinterpret_cast<uint32_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU64:
    return reference::device::BlockCompareEqual<uint64_t>(
      reinterpret_cast<uint64_t const *>(ptr_A), 
      reinterpret_cast<uint64_t const *>(ptr_B), 
      capacity);

  default:
    throw std::runtime_error("Unsupported numeric type");
  }
}

/// Returns true if two blocks have approximately the same value
bool DeviceAllocation::block_compare_relatively_equal(
  library::NumericTypeID numeric_type, 
  void const *ptr_A, 
  void const *ptr_B, 
  size_t capacity,
  double epsilon,
  double nonzero_floor) {

  switch (numeric_type) {
  case library::NumericTypeID::kF16:
    return reference::device::BlockCompareRelativelyEqual<half_t>(
      reinterpret_cast<half_t const *>(ptr_A), 
      reinterpret_cast<half_t const *>(ptr_B),
      capacity, 
      static_cast<half_t>(epsilon), 
      static_cast<half_t>(nonzero_floor));
    
  case library::NumericTypeID::kBF16:
    return reference::device::BlockCompareRelativelyEqual<bfloat16_t>(
      reinterpret_cast<bfloat16_t const *>(ptr_A), 
      reinterpret_cast<bfloat16_t const *>(ptr_B),
      capacity, 
      static_cast<bfloat16_t>(epsilon), 
      static_cast<bfloat16_t>(nonzero_floor));

  case library::NumericTypeID::kTF32:
    return reference::device::BlockCompareRelativelyEqual<tfloat32_t>(
      reinterpret_cast<tfloat32_t const *>(ptr_A), 
      reinterpret_cast<tfloat32_t const *>(ptr_B),
      capacity, 
      static_cast<tfloat32_t>(epsilon), 
      static_cast<tfloat32_t>(nonzero_floor));

  case library::NumericTypeID::kF32:
    return reference::device::BlockCompareRelativelyEqual<float>(
      reinterpret_cast<float const *>(ptr_A), 
      reinterpret_cast<float const *>(ptr_B),
      capacity, 
      static_cast<float>(epsilon), 
      static_cast<float>(nonzero_floor));

  case library::NumericTypeID::kF64:
    return reference::device::BlockCompareRelativelyEqual<double>(
      reinterpret_cast<double const *>(ptr_A), 
      reinterpret_cast<double const *>(ptr_B),
      capacity, 
      static_cast<double>(epsilon), 
      static_cast<double>(nonzero_floor));
  
  case library::NumericTypeID::kS2:
    return reference::device::BlockCompareRelativelyEqual<int2b_t>(
      reinterpret_cast<int2b_t const *>(ptr_A), 
      reinterpret_cast<int2b_t const *>(ptr_B),
      capacity, 
      static_cast<int2b_t>(epsilon), 
      static_cast<int2b_t>(nonzero_floor));
  
  case library::NumericTypeID::kS4:
    return reference::device::BlockCompareRelativelyEqual<int4b_t>(
      reinterpret_cast<int4b_t const *>(ptr_A), 
      reinterpret_cast<int4b_t const *>(ptr_B),
      capacity, 
      static_cast<int4b_t>(epsilon), 
      static_cast<int4b_t>(nonzero_floor));

  case library::NumericTypeID::kS8:
    return reference::device::BlockCompareRelativelyEqual<int8_t>(
      reinterpret_cast<int8_t const *>(ptr_A), 
      reinterpret_cast<int8_t const *>(ptr_B),
      capacity, 
      static_cast<int8_t>(epsilon), 
      static_cast<int8_t>(nonzero_floor));

  case library::NumericTypeID::kS16:
    return reference::device::BlockCompareRelativelyEqual<int16_t>(
      reinterpret_cast<int16_t const *>(ptr_A), 
      reinterpret_cast<int16_t const *>(ptr_B),
      capacity, 
      static_cast<int16_t>(epsilon), 
      static_cast<int16_t>(nonzero_floor));

  case library::NumericTypeID::kS32:
    return reference::device::BlockCompareRelativelyEqual<int32_t>(
      reinterpret_cast<int32_t const *>(ptr_A), 
      reinterpret_cast<int32_t const *>(ptr_B),
      capacity, 
      static_cast<int32_t>(epsilon), 
      static_cast<int32_t>(nonzero_floor));

  case library::NumericTypeID::kS64:
    return reference::device::BlockCompareRelativelyEqual<int64_t>(
      reinterpret_cast<int64_t const *>(ptr_A), 
      reinterpret_cast<int64_t const *>(ptr_B),
      capacity, 
      static_cast<int64_t>(epsilon), 
      static_cast<int64_t>(nonzero_floor));
  
  case library::NumericTypeID::kB1:
    return reference::device::BlockCompareRelativelyEqual<uint1b_t>(
      reinterpret_cast<uint1b_t const *>(ptr_A), 
      reinterpret_cast<uint1b_t const *>(ptr_B),
      capacity, 
      static_cast<uint1b_t>(epsilon), 
      static_cast<uint1b_t>(nonzero_floor));

  case library::NumericTypeID::kU2:
    return reference::device::BlockCompareRelativelyEqual<uint2b_t>(
      reinterpret_cast<uint2b_t const *>(ptr_A), 
      reinterpret_cast<uint2b_t const *>(ptr_B),
      capacity, 
      static_cast<uint2b_t>(epsilon), 
      static_cast<uint2b_t>(nonzero_floor));

  case library::NumericTypeID::kU4:
    return reference::device::BlockCompareRelativelyEqual<uint4b_t>(
      reinterpret_cast<uint4b_t const *>(ptr_A), 
      reinterpret_cast<uint4b_t const *>(ptr_B),
      capacity, 
      static_cast<uint4b_t>(epsilon), 
      static_cast<uint4b_t>(nonzero_floor));

  case library::NumericTypeID::kU8:
    return reference::device::BlockCompareRelativelyEqual<uint8_t>(
      reinterpret_cast<uint8_t const *>(ptr_A), 
      reinterpret_cast<uint8_t const *>(ptr_B),
      capacity, 
      static_cast<uint8_t>(epsilon), 
      static_cast<uint8_t>(nonzero_floor));

  case library::NumericTypeID::kU16:
    return reference::device::BlockCompareRelativelyEqual<uint16_t>(
      reinterpret_cast<uint16_t const *>(ptr_A), 
      reinterpret_cast<uint16_t const *>(ptr_B),
      capacity, 
      static_cast<uint16_t>(epsilon), 
      static_cast<uint16_t>(nonzero_floor));

  case library::NumericTypeID::kU32:
    return reference::device::BlockCompareRelativelyEqual<uint32_t>(
      reinterpret_cast<uint32_t const *>(ptr_A), 
      reinterpret_cast<uint32_t const *>(ptr_B),
      capacity, 
      static_cast<uint32_t>(epsilon), 
      static_cast<uint32_t>(nonzero_floor));

  case library::NumericTypeID::kU64:
    return reference::device::BlockCompareRelativelyEqual<uint64_t>(
      reinterpret_cast<uint64_t const *>(ptr_A), 
      reinterpret_cast<uint64_t const *>(ptr_B),
      capacity, 
      static_cast<uint64_t>(epsilon), 
      static_cast<uint64_t>(nonzero_floor));

  // No relatively equal comparison for complex numbers.
  //
  // As a simplification, we can require bitwise equality. This avoids false positives.
  // (i.e. "pass" really means passing. "Fail" may not actually mean failure given appropriate epsilon.)
  //
  case library::NumericTypeID::kCF16:
    return reference::device::BlockCompareEqual<cutlass::complex<half_t> >(
      reinterpret_cast<complex<half_t> const *>(ptr_A),
      reinterpret_cast<complex<half_t> const *>(ptr_B),
      capacity);

  case library::NumericTypeID::kCF32:
    return reference::device::BlockCompareEqual<cutlass::complex<float> >(
      reinterpret_cast<complex<float> const *>(ptr_A),
      reinterpret_cast<complex<float> const *>(ptr_B),
      capacity);
  
  case library::NumericTypeID::kCF64:
    return reference::device::BlockCompareEqual<cutlass::complex<double> >(
      reinterpret_cast<complex<double> const *>(ptr_A),
      reinterpret_cast<complex<double> const *>(ptr_B),
      capacity);

  default:
    {
      throw std::runtime_error(std::string("Unsupported numeric type: ") + to_string(numeric_type));
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Permits copying dynamic vectors into static-length vectors 
template <typename TensorCoord, int Rank>
struct vector_to_coord {
  
  vector_to_coord(TensorCoord &coord, std::vector<int> const &vec) {

    coord[Rank - 1] = vec.at(Rank - 1);
    
    if (Rank > 1) {
      vector_to_coord<TensorCoord, Rank - 1>(coord, vec);
    }
  }

  vector_to_coord(TensorCoord &coord, std::vector<int64_t> const &vec) {

    coord[Rank - 1] = (int)vec.at(Rank - 1);
    
    if (Rank > 1) {
      vector_to_coord<TensorCoord, Rank - 1>(coord, vec);
    }
  }
};

/// Permits copying dynamic vectors into static-length vectors 
template <typename TensorCoord>
struct vector_to_coord<TensorCoord, 1> {
  
  vector_to_coord(TensorCoord &coord, std::vector<int> const &vec) {

    coord[0] = vec.at(0);
  }

  vector_to_coord(TensorCoord &coord, std::vector<int64_t> const &vec) {

    coord[0] = (int)vec.at(0);
  }
};

/// Permits copying dynamic vectors into static-length vectors 
template <typename TensorCoord>
struct vector_to_coord<TensorCoord, 0> {
  
  vector_to_coord(TensorCoord &coord, std::vector<int> const &vec) {

  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Element, typename Layout>
static void write_tensor_csv_static_tensor_view(
  std::ostream &out, 
  DeviceAllocation &allocation) {

  Coord<Layout::kRank> extent;
  Coord<Layout::kStrideRank, typename Layout::Stride::Index> stride;

  if (allocation.extent().size() != Layout::kRank) {
    throw std::runtime_error("Allocation extent has invalid rank");
  }

  if (allocation.stride().size() != Layout::kStrideRank) {
    throw std::runtime_error("Allocation stride has invalid rank");
  }

  vector_to_coord<Coord<Layout::kRank>, Layout::kRank>(extent, allocation.extent());
  vector_to_coord<Coord<Layout::kStrideRank, typename Layout::Stride::Index>, 
                        Layout::kStrideRank>(stride, allocation.stride());

  Layout layout(stride);
  HostTensor<Element, Layout> host_tensor(extent, layout, false);

  if (host_tensor.capacity() != allocation.batch_stride()) {
    throw std::runtime_error("Unexpected capacity to equal.");
  }

  host_tensor.copy_in_device_to_host(
    static_cast<Element const *>(allocation.data()), 
    allocation.batch_stride());

  TensorViewWrite(out, host_tensor.host_view());

  out << "\n\n";
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
static void write_tensor_csv_static_type(
  std::ostream &out, 
  DeviceAllocation &allocation) {

  switch (allocation.layout()) {
    case library::LayoutTypeID::kRowMajor:
      write_tensor_csv_static_tensor_view<T, layout::RowMajor>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajor:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajor>(out, allocation);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK2:
      write_tensor_csv_static_tensor_view<T, layout::RowMajorInterleaved<2>>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK2:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajorInterleaved<2>>(out, allocation);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK4:
      write_tensor_csv_static_tensor_view<T, layout::RowMajorInterleaved<4>>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK4:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajorInterleaved<4>>(out, allocation);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK16:
      write_tensor_csv_static_tensor_view<T, layout::RowMajorInterleaved<16>>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK16:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajorInterleaved<16>>(out, allocation);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK32:
      write_tensor_csv_static_tensor_view<T, layout::RowMajorInterleaved<32>>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK32:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajorInterleaved<32>>(out, allocation);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK64:
      write_tensor_csv_static_tensor_view<T, layout::RowMajorInterleaved<64>>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK64:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajorInterleaved<64>>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorNHWC:
      write_tensor_csv_static_tensor_view<T, layout::TensorNHWC>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorNDHWC:
      write_tensor_csv_static_tensor_view<T, layout::TensorNDHWC>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorNC32HW32:
      write_tensor_csv_static_tensor_view<T, layout::TensorNCxHWx<32>>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorNC64HW64:
      write_tensor_csv_static_tensor_view<T, layout::TensorNCxHWx<64>>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorC32RSK32:
      write_tensor_csv_static_tensor_view<T, layout::TensorCxRSKx<32>>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorC64RSK64:
      write_tensor_csv_static_tensor_view<T, layout::TensorCxRSKx<64>>(out, allocation);
      break;
    default:
      throw std::runtime_error("Unhandled layout");
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Writes a tensor to csv 
void DeviceAllocation::write_tensor_csv(
  std::ostream &out) {

  switch (this->type()) {
  case library::NumericTypeID::kF16:
    write_tensor_csv_static_type<half_t>(out, *this);
    break;
    
  case library::NumericTypeID::kBF16:
    write_tensor_csv_static_type<bfloat16_t>(out, *this);
    break;

  case library::NumericTypeID::kTF32:
    write_tensor_csv_static_type<tfloat32_t>(out, *this);
    break;

  case library::NumericTypeID::kF32:
    write_tensor_csv_static_type<float>(out, *this);
    break;

  case library::NumericTypeID::kF64:
    write_tensor_csv_static_type<double>(out, *this);
    break;
  
  case library::NumericTypeID::kS2:
    write_tensor_csv_static_type<int2b_t>(out, *this);
    break;

  case library::NumericTypeID::kS4:
    write_tensor_csv_static_type<int4b_t>(out, *this);
    break;

  case library::NumericTypeID::kS8:
    write_tensor_csv_static_type<int8_t>(out, *this);
    break;

  case library::NumericTypeID::kS16:
    write_tensor_csv_static_type<int16_t>(out, *this);
    break;

  case library::NumericTypeID::kS32:
    write_tensor_csv_static_type<int32_t>(out, *this);
    break;

  case library::NumericTypeID::kS64:
    write_tensor_csv_static_type<int64_t>(out, *this);
    break;
  
  case library::NumericTypeID::kB1:
    write_tensor_csv_static_type<uint1b_t>(out, *this);
    break;

  case library::NumericTypeID::kU2:
    write_tensor_csv_static_type<uint2b_t>(out, *this);
    break;

  case library::NumericTypeID::kU4:
    write_tensor_csv_static_type<uint4b_t>(out, *this);
    break;

  case library::NumericTypeID::kU8:
    write_tensor_csv_static_type<uint8_t>(out, *this);
    break;

  case library::NumericTypeID::kU16:
    write_tensor_csv_static_type<uint16_t>(out, *this);
    break;

  case library::NumericTypeID::kU32:
    write_tensor_csv_static_type<uint32_t>(out, *this);
    break;

  case library::NumericTypeID::kU64:
    write_tensor_csv_static_type<uint64_t>(out, *this);
    break;
  
  case library::NumericTypeID::kCF16:
    write_tensor_csv_static_type<cutlass::complex<half_t> >(out, *this);
    break;

  case library::NumericTypeID::kCF32:
    write_tensor_csv_static_type<cutlass::complex<float> >(out, *this);
    break;

  case library::NumericTypeID::kCF64:
    write_tensor_csv_static_type<cutlass::complex<double> >(out, *this);
    break;

  default:
    throw std::runtime_error("Unsupported numeric type");
  }
}

template <typename Element, typename Layout>
static void tensor_fill_tensor_view(DeviceAllocation &allocation, Element val = Element()) {
  Coord<Layout::kRank> extent;
  Coord<Layout::kStrideRank, typename Layout::LongIndex> stride;

  if (allocation.extent().size() != Layout::kRank) {
    throw std::runtime_error("Allocation extent has invalid rank");
  }

  if (allocation.stride().size() != Layout::kStrideRank) {
    throw std::runtime_error("Allocation stride has invalid rank");
  }

  vector_to_coord<Coord<Layout::kRank>, Layout::kRank>(extent, allocation.extent());
  vector_to_coord<Coord<Layout::kStrideRank, typename Layout::LongIndex>, 
                        Layout::kStrideRank>(stride, allocation.stride());

  TensorView<Element, Layout> view(
    static_cast<Element *>(allocation.data()),
    Layout(stride),
    extent
  );


  cutlass::reference::device::TensorFill<Element, Layout>(
    view,
    val
  );
}

template <typename Element>
static void tensor_fill(DeviceAllocation &allocation, Element val = Element()) {
  switch (allocation.layout()) {
    case library::LayoutTypeID::kRowMajor:
      tensor_fill_tensor_view<Element, layout::RowMajor>(allocation, val);
      break;
    case library::LayoutTypeID::kColumnMajor:
      tensor_fill_tensor_view<Element, layout::ColumnMajor>(allocation, val);
      break;
    case library::LayoutTypeID::kTensorNHWC:
      tensor_fill_tensor_view<Element, layout::TensorNHWC>(allocation, val);
      break;
    case library::LayoutTypeID::kTensorNDHWC:
      tensor_fill_tensor_view<Element, layout::TensorNDHWC>(allocation, val);
      break;
    case library::LayoutTypeID::kTensorNC32HW32:
      tensor_fill_tensor_view<Element, layout::TensorNCxHWx<32>>(allocation, val);
      break;
    case library::LayoutTypeID::kTensorNC64HW64:
      tensor_fill_tensor_view<Element, layout::TensorNCxHWx<64>>(allocation, val);
      break;
    case library::LayoutTypeID::kTensorC32RSK32:
      tensor_fill_tensor_view<Element, layout::TensorCxRSKx<32>>(allocation, val);
      break;
    case library::LayoutTypeID::kTensorC64RSK64:
      tensor_fill_tensor_view<Element, layout::TensorCxRSKx<64>>(allocation, val);
      break;
    default:
    throw std::runtime_error("Unsupported layout");
      break;
  }
}

/// Fills a tensor uniformly with a value (most frequently used to clear the tensor)
void DeviceAllocation::fill(double val = 0.0) {

  switch (this->type()) {
  case library::NumericTypeID::kF16:
    tensor_fill<half_t>(*this, static_cast<half_t>(val));
    break;

  case library::NumericTypeID::kBF16:
    tensor_fill<bfloat16_t>(*this, static_cast<bfloat16_t>(val));
    break;

  case library::NumericTypeID::kTF32:
    tensor_fill<tfloat32_t>(*this, static_cast<tfloat32_t>(val));
    break;

  case library::NumericTypeID::kF32:
    tensor_fill<float>(*this, static_cast<float>(val));
    break;

  case library::NumericTypeID::kF64:
    tensor_fill<double>(*this, static_cast<double>(val));
    break;

  case library::NumericTypeID::kS2:
    tensor_fill<int2b_t>(*this, static_cast<int2b_t>(val));
    break;

  case library::NumericTypeID::kS4:
    tensor_fill<int4b_t>(*this, static_cast<int4b_t>(val));
    break;

  case library::NumericTypeID::kS8:
    tensor_fill<int8_t>(*this, static_cast<int8_t>(val));
    break;

  case library::NumericTypeID::kS16:
    tensor_fill<int16_t>(*this, static_cast<int16_t>(val));
    break;

  case library::NumericTypeID::kS32:
    tensor_fill<int32_t>(*this, static_cast<int32_t>(val));
    break;

  case library::NumericTypeID::kS64:
    tensor_fill<int64_t>(*this, static_cast<int64_t>(val));
    break;

  case library::NumericTypeID::kB1:
    tensor_fill<uint1b_t>(*this, static_cast<uint1b_t>(val));
    break;

  case library::NumericTypeID::kU2:
    tensor_fill<uint2b_t>(*this, static_cast<uint2b_t>(val));
    break;

  case library::NumericTypeID::kU4:
    tensor_fill<uint4b_t>(*this, static_cast<uint4b_t>(val));
    break;

  case library::NumericTypeID::kU8:
    tensor_fill<uint8_t>(*this, static_cast<uint8_t>(val));
    break;

  case library::NumericTypeID::kU16:
    tensor_fill<uint16_t>(*this, static_cast<uint16_t>(val));
    break;

  case library::NumericTypeID::kU32:
    tensor_fill<uint32_t>(*this, static_cast<uint32_t>(val));
    break;

  case library::NumericTypeID::kU64:
    tensor_fill<uint64_t>(*this, static_cast<uint64_t>(val));
    break;

  case library::NumericTypeID::kCF16:
    tensor_fill<cutlass::complex<half_t> >(*this, from_real<half_t>(val));
    break;

  case library::NumericTypeID::kCF32:
    tensor_fill<cutlass::complex<float> >(*this, from_real<float>(val));
    break;

  case library::NumericTypeID::kCF64:
    tensor_fill<cutlass::complex<double> >(*this, from_real<double>(val));
    break;

  default:
    throw std::runtime_error("Unsupported numeric type");
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass
