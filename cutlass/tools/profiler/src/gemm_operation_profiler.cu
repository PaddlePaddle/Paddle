#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <iostream>
#include <stdexcept>
#include <iomanip>
#include <ios>

#include "cutlass/core_io.h"

#include "cublas_helpers.h"
#include "gemm_operation_profiler.h"
#include "gpu_timer.h"

#include "cutlass/library/singleton.h"
#include "cutlass/library/library.h"
#include "cutlass/library/handle.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {


/////////////////////////////////////////////////////////////////////////////////////////////////

/// Ctor
GemmOperationProfiler::GemmOperationProfiler(Options const &options): 
  OperationProfiler(
    options,
    library::OperationKind::kGemm,
    {
      {ArgumentTypeID::kEnumerated, {"gemm_kind"}, "Variant of GEMM (gemm, batched, array, universal, planar_complex, planar_complex_array)"},
      {ArgumentTypeID::kEnumerated, {"split_k_mode"}, "Variant of split K mode(serial, parallel)"},
      {ArgumentTypeID::kInteger, {"m", "problem-size::m"}, "M dimension of the GEMM problem space"},
      {ArgumentTypeID::kInteger, {"n", "problem-size::n"}, "N dimension of the GEMM problem space"},
      {ArgumentTypeID::kInteger, {"k", "problem-size::k"}, "K dimension of the GEMM problem space"},
      {ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
      {ArgumentTypeID::kTensor, {"B"}, "Tensor storing the B operand"},
      {ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
      {ArgumentTypeID::kScalar, {"alpha", "epilogue::alpha"}, "Epilogue scalar alpha"},
      {ArgumentTypeID::kScalar, {"beta", "epilogue::beta"}, "Epilogue scalar beta"},
      {ArgumentTypeID::kInteger, {"split_k_slices", "split-k-slices"}, "Number of partitions of K dimension"},
      {ArgumentTypeID::kInteger, {"batch_count", "batch-count"}, "Number of GEMMs computed in one batch"},
    },
    { library::Provider::kCUBLAS}
  ) {

  description_ = "      General matrix-matrix product. D = alpha * A*B + beta * C";
}

/// Destructor
GemmOperationProfiler::~GemmOperationProfiler() {

}

/// Prints usage statement for the math function
void GemmOperationProfiler::print_usage(std::ostream &out) const {
  out << "GEMM" << "\n\n";

  OperationProfiler::print_usage(out);
}

/// Prints examples
void GemmOperationProfiler::print_examples(std::ostream &out) const {

  out << "\nExamples:\n\n"
    << "Profile a particular problem size:\n"
    << "  $ cutlass_profiler --operation=Gemm --m=1024 --n=1024 --k=128\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=Gemm --m=1024:4096:256 --n=1024:4096:256 --k=128:8192:128 --beta=0,1,2.5\n\n"

    << "Schmoo over accumulator types:\n"
    << "  $ cutlass_profiler --operation=Gemm --accumulator-type=f16,f32\n\n"

    << "Run when A is f16 with column-major and B is any datatype with row-major (For column major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=Gemm --A=f16:column --B=*:row\n\n"

    << "Profile a particular problem size with split K and paralell reduction:\n"
    << "  $ cutlass_profiler --operation=Gemm --split_k_mode=parallel --split_k_slices=2 --m=1024 --n=1024 --k=128\n\n"

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=sequential,start:0,delta:1\n\n"

    << "Run a kernel with cta tile size of 256x128x32 and save workspace if results are incorrect (note that --cta-tile::k=32 is default cta-tile size):\n"
    << " $ cutlass_profiler --operation=Gemm --cta_m=256 --cta_n=128  --cta_k=32 --save-workspace=incorrect\n\n"
    
    << "Test your changes to gemm kernels with a quick functional test and save results in functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=Gemm \\ \n"
    << "   --m=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --n=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --k=8,16,32,64,128,256,288,384,504,512,520 \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#if 0
// used this for debugging
static std::string byte_string(std::vector<uint8_t> const &bytes) {
  std::stringstream ss;

  ss << "0x";

  for (size_t idx = bytes.size(); idx > 0; --idx) {
    ss << std::hex << std::setw(2) << std::setfill('0') << uint32_t(bytes.at(idx - 1));
  }

  return ss.str();
}
#endif

Status GemmOperationProfiler::GemmProblem::parse(
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {
    
  this->mode = library::GemmUniversalMode::kGemm;
  
  if (!arg_as_int(this->m, "m", problem_space, problem)) {
    // default value
    this->m = 1024;
  }

  if (!arg_as_int(this->n, "n", problem_space, problem)) {
    // default value
    this->n = 1024;
  }
  
  if (!arg_as_int(this->k, "k", problem_space, problem)) {
    // default value
    this->k = 1024;
  }

  if (!arg_as_SplitKModeID(this->split_k_mode, "split_k_mode", problem_space, problem)) {
    // defualt value
    this->split_k_mode = library::SplitKMode::kSerial;
  }
  
  this->mode = library::GemmUniversalMode::kGemm;
  if(this->split_k_mode == library::SplitKMode::kParallel) {
    this->mode = library::GemmUniversalMode::kGemmSplitKParallel;
  }

  if (!arg_as_int(this->split_k_slices, "split_k_slices", problem_space, problem)) {
    // default value
    this->split_k_slices = 1;
  }
  
  if (!arg_as_int(this->batch_count, "batch_count", problem_space, problem)) {
    // default value
    this->batch_count = 1;
  } else if (this->batch_count > 1) {
    this->mode = library::GemmUniversalMode::kBatched;
  }

  if (this->split_k_slices > 1 && this->batch_count > 1) {
    // At least one of these must be one
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.B, "B", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
    this->alpha, 
    operation_desc.element_epilogue, 
    "alpha", 
    problem_space, 
    problem)) {

    if (!cast_from_double(this->alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }
  
  if (!arg_as_scalar(
    this->beta, 
    operation_desc.element_epilogue, 
    "beta", 
    problem_space, 
    problem)) {
    
    if (!cast_from_double(this->beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }

  this->lda = DeviceAllocation::get_packed_layout(
    operation_desc.A.layout, {int(this->m), int(this->k)}).front();

  this->ldb = DeviceAllocation::get_packed_layout(
    operation_desc.B.layout, {int(this->k), int(this->n)}).front();

  this->ldc = DeviceAllocation::get_packed_layout(
    operation_desc.C.layout, {int(this->m), int(this->n)}).front();

  return Status::kSuccess;
}

/// Total number of bytes loaded
int64_t GemmOperationProfiler::GemmProblem::bytes(library::GemmDescription const &operation_desc) const {
  // Input bytes read and Output bytes written for the gemm problem
  int64_t bytes =
    int64_t(library::sizeof_bits(operation_desc.A.element) * m / 8) * k +
    int64_t(library::sizeof_bits(operation_desc.B.element) * n / 8) * k +
    int64_t(library::sizeof_bits(operation_desc.C.element) * m / 8) * n;

  // Set is_beta_zero true if beta is zero
  bool is_beta_zero = std::all_of(beta.begin(), beta.end(), [](uint8_t i) { return i==0; });

  // Output bytes read for the gemm problem for non-zero beta values
  if (!is_beta_zero) {
    bytes += int64_t(library::sizeof_bits(operation_desc.C.element) * m / 8) * n;
  }

  bytes *= batch_count;

  return bytes;
}

/// Total number of flops computed
int64_t GemmOperationProfiler::GemmProblem::flops(library::GemmDescription const &operation_desc) const {
  int64_t flops_ = (int64_t(m) * n * k + m * n) * 2 * batch_count;

  // complex-valued support
  switch (operation_desc.tile_description.math_instruction.math_operation) {
  case library::MathOperationID::kMultiplyAddComplex:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddComplexFastF32:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddGaussianComplex:
    flops_ *= 3;
    break;

  default: break;
  }

  return flops_;
}


/// Initializes a performance result
void GemmOperationProfiler::GemmProblem::initialize_result(
  PerformanceResult &result,
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.arguments.resize(problem_space.rank());

  set_argument(result, "gemm_kind", problem_space, library::to_string(operation_desc.gemm_kind));

  set_argument(result, "split_k_mode", problem_space, library::to_string(split_k_mode));

  set_argument(result, "A", problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" + library::to_string(operation_desc.A.layout));

  set_argument(result, "B", problem_space,
    std::string(library::to_string(operation_desc.B.element)) + ":" + library::to_string(operation_desc.B.layout));

  set_argument(result, "C", problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" + library::to_string(operation_desc.C.layout));

  set_argument(result, "m", problem_space, m);
  set_argument(result, "n", problem_space, n);
  set_argument(result, "k", problem_space, k);

  set_argument(result, "split_k_slices", problem_space, split_k_slices);
  set_argument(result, "batch_count", problem_space, batch_count);

  set_argument(result, "alpha", problem_space,
    library::lexical_cast(alpha, operation_desc.element_epilogue));

  set_argument(result, "beta", problem_space,
    library::lexical_cast(beta, operation_desc.element_epilogue));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Extracts the problem dimensions
Status GemmOperationProfiler::initialize_configuration(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::GemmDescription const &operation_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  if (operation_desc.gemm_kind != library::GemmKind::kUniversal) {
    return Status::kErrorInvalidProblem;
  }

  Status status = problem_.parse(operation_desc, problem_space, problem);

  if (status != Status::kSuccess) {
    return status;
  }

  gemm_workspace_.configuration.mode = problem_.mode;
  gemm_workspace_.configuration.problem_size.m() = int(problem_.m);
  gemm_workspace_.configuration.problem_size.n() = int(problem_.n);
  gemm_workspace_.configuration.problem_size.k() = int(problem_.k);
  gemm_workspace_.configuration.lda = problem_.lda;
  gemm_workspace_.configuration.ldb = problem_.ldb;
  gemm_workspace_.configuration.ldc = problem_.ldc;
  gemm_workspace_.configuration.ldd = problem_.ldc;

  if (problem_.mode == library::GemmUniversalMode::kBatched) {
    gemm_workspace_.configuration.batch_count = problem_.batch_count;
  }
  else {
    gemm_workspace_.configuration.batch_count = problem_.split_k_slices;
  }

  gemm_workspace_.arguments.A = nullptr;
  gemm_workspace_.arguments.B = nullptr;
  gemm_workspace_.arguments.C = nullptr;
  gemm_workspace_.arguments.D = nullptr;
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  // initialize reduction operation for parallel splitKMode
  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!initialize_reduction_configuration_(operation, problem)) {
      return Status::kErrorInternal;
    }
  }

  initialize_result_(this->model_result_, options, operation_desc, problem_space);
  
  return operation->can_implement(&gemm_workspace_.configuration, &gemm_workspace_.arguments);
}

/// Initializes the performance result
void GemmOperationProfiler::initialize_result_(
  PerformanceResult &result,
  Options const &options,  
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.provider = library::Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;

  problem_.initialize_result(result, operation_desc, problem_space);

  OperationProfiler::initialize_result_(result, operation_desc, problem_space);

  result.bytes = problem_.bytes(operation_desc);
  result.flops = problem_.flops(operation_desc);
  result.runtime = 0;

}

/// Initialize redution problem dimentions and library::Operation
bool GemmOperationProfiler::initialize_reduction_configuration_(
  library::Operation const *operation,
  ProblemSpace::Problem const &problem) {
  library::GemmDescription const &gemm_desc =
    static_cast<library::GemmDescription const&>(operation->description());

  if (!cast_from_double(problem_.alpha_one, gemm_desc.element_epilogue, 1)) {
    return false;
  }

  if (!cast_from_double(problem_.beta_zero, gemm_desc.element_epilogue, 0)) {
    return false;
  }

  /// initialize library::ReductionConfiguration
  gemm_workspace_.reduction_configuration.problem_size      = gemm::GemmCoord(int(problem_.n), int(problem_.m), int(problem_.k)).mn();
  gemm_workspace_.reduction_configuration.partitions        = int(problem_.split_k_slices);
  gemm_workspace_.reduction_configuration.partition_stride  = gemm::GemmCoord(int(problem_.n), int(problem_.m), int(problem_.k)).mn().product();
  gemm_workspace_.reduction_configuration.ldw               = problem_.ldc;
  gemm_workspace_.reduction_configuration.lds               = problem_.ldc;
  gemm_workspace_.reduction_configuration.ldd               = problem_.ldc;

  // find reduction operation
  library::ReductionFunctionalKey reduction_key(
    library::Provider::kCUTLASS,
    gemm_desc.tile_description.math_instruction.element_accumulator,    // element workspace
    gemm_desc.tile_description.math_instruction.element_accumulator,    // element accumulator
    gemm_desc.C.element,                                                // element output
    gemm_desc.element_epilogue                                          // element coumpute
  );

  auto reduction_it = library::Singleton::get().operation_table.reduction_operations.find(reduction_key);
 
  if (reduction_it == library::Singleton::get().operation_table.reduction_operations.end()) {
    return false;
  }

  // initialize reduction operation required for parallel split-k operator
  reduction_op_ = reduction_it->second;

  // reduction operation found and initialized
  return true;
}

/// Initializes workspace
Status GemmOperationProfiler::initialize_workspace(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::Operation const* underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      return Status::kErrorNotSupported;
    }
  }

  library::GemmDescription const &operation_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  // Compute the number of copies of the problem to avoid L2 camping.
  if (!options.profiling.workspace_count) {
    int64_t bytes = problem_.bytes(operation_desc);
    if (bytes < 3 * int64_t(options.device.properties.l2CacheSize)) {
      gemm_workspace_.problem_count = 
        1 + int((3 * int64_t(options.device.properties.l2CacheSize)) / bytes);
    }
    else {
      gemm_workspace_.problem_count = 1;
    }
  }
  else {
    gemm_workspace_.problem_count = options.profiling.workspace_count;
  }

  if (options.execution_mode != ExecutionMode::kDryRun) {

    gemm_workspace_.A = device_context.allocate_tensor(
      options,
      "A",
      operation_desc.A.element,
      operation_desc.A.layout,
      {int(problem_.m), int(problem_.k)},
      {int(problem_.lda)},
      problem_.batch_count * gemm_workspace_.problem_count
    );

    gemm_workspace_.B = device_context.allocate_tensor(
      options,
      "B",
      operation_desc.B.element,
      operation_desc.B.layout,
      {int(problem_.k), int(problem_.n)},
      {int(problem_.ldb)},
      problem_.batch_count * gemm_workspace_.problem_count
    );

    gemm_workspace_.C = device_context.allocate_tensor(
      options,
      "C",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)},
      problem_.batch_count * gemm_workspace_.problem_count
    );

    gemm_workspace_.Computed = device_context.allocate_tensor(
      "D",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)},
      problem_.batch_count * gemm_workspace_.problem_count
    );

    gemm_workspace_.Reference = device_context.allocate_tensor(
      "Reference",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)},
      problem_.batch_count * gemm_workspace_.problem_count
    );

    gemm_workspace_.Reference->copy_from_device(gemm_workspace_.C->data());

    gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
    gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
    gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.C->batch_stride();
    gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Computed->batch_stride();
  }

  //
  // Initialize the CUTLASS operation
  //
  Status status = Status::kSuccess;

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    if (options.execution_mode != ExecutionMode::kDryRun) {

      uint64_t workspace_size = underlying_operation->get_host_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.host_workspace.resize(workspace_size, 0);

      workspace_size = underlying_operation->get_device_workspace_size(&gemm_workspace_.configuration,
                                                            &gemm_workspace_.arguments);
      gemm_workspace_.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

      status = underlying_operation->initialize(
        &gemm_workspace_.configuration,
        gemm_workspace_.host_workspace.data(),
        gemm_workspace_.device_workspace.data());

      if (status != Status::kSuccess) {
        return status;
      }

      if (problem_.split_k_mode == library::SplitKMode::kParallel) {
        workspace_size = reduction_op_->get_host_workspace_size(&gemm_workspace_.reduction_configuration);
        gemm_workspace_.reduction_host_workspace.resize(workspace_size, 0);

        status = reduction_op_->initialize(
          &gemm_workspace_.reduction_configuration,
          gemm_workspace_.reduction_host_workspace.data(),
          nullptr);

        if (status != Status::kSuccess) {
          return status;
        }
      }
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //
    results_.push_back(model_result_);
    results_.back().provider = library::Provider::kCUTLASS;
    results_.back().op_kind = library::OperationKind::kGemm;
    results_.back().disposition = Disposition::kNotRun;

    for(auto provider : verification_providers_) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
    }
  }

  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GemmOperationProfiler::verify_cutlass(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (!options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  // Initialize structure containing GEMM arguments
  gemm_workspace_.arguments.A = gemm_workspace_.A->data();
  gemm_workspace_.arguments.B = gemm_workspace_.B->data();
  gemm_workspace_.arguments.C = gemm_workspace_.C->data();
  gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;
  gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
  gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
  gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.C->batch_stride();
  gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Computed->batch_stride();

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    gemm_workspace_.arguments.D                       = gemm_workspace_.device_workspace.data();
    gemm_workspace_.arguments.alpha                   = problem_.alpha_one.data();
    gemm_workspace_.arguments.beta                    = problem_.beta_zero.data();

    gemm_workspace_.reduction_arguments.workspace     = gemm_workspace_.device_workspace.data();
    gemm_workspace_.reduction_arguments.source        = gemm_workspace_.C->data();
    gemm_workspace_.reduction_arguments.destination   = gemm_workspace_.Computed->data();
    gemm_workspace_.reduction_arguments.alpha         = problem_.alpha.data();
    gemm_workspace_.reduction_arguments.beta          = problem_.beta.data();
    gemm_workspace_.reduction_arguments.pointer_mode  = library::ScalarPointerMode::kHost;
  }

  //
  // Run the CUTLASS operation
  //

 // initialize gemm underlying operation to handle parallel reduction
  library::Operation const * underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      results_.back().disposition = Disposition::kFailed;
      return false;
    }
  }

  results_.back().status = underlying_operation->run(
    &gemm_workspace_.arguments, 
    gemm_workspace_.host_workspace.data(),
    gemm_workspace_.device_workspace.data());

  if (results_.back().status != Status::kSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // Run parallel reduction kernel for parallel split_k_mode
  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    results_.back().status = reduction_op_->run(
      &gemm_workspace_.reduction_arguments,
      gemm_workspace_.reduction_host_workspace.data(),
      nullptr);

    if (results_.back().status != Status::kSuccess) {
      results_.back().disposition = Disposition::kFailed;
      return false;
    }
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // CUTLASS op ran the but not yet verified against any verification provider
  results_.back().disposition = Disposition::kNotVerified;

  //
  // Run verification providers
  //

  if (options.verification.enabled) {

#if CUTLASS_ENABLE_CUBLAS
    if (options.verification.provider_enabled(library::Provider::kCUBLAS)) {

      // Guard against unsupported cases
      auto const & gemm_desc = static_cast<library::GemmDescription const &>(operation->description());

      if (cublas_satisfies(gemm_desc) == Status::kSuccess) {

        // call cublas verification if supported
        verify_with_cublas_(
          options,
          report,
          device_context,
          operation,
          problem_space,
          problem);
        }

      else {
        // set verification map for cublas to not supported
        results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotSupported;
      }
    }
#endif // #if CUTLASS_ENABLE_CUBLAS

    verify_with_reference_(options, report, device_context, operation, problem_space, problem);
    
    // Update disposition to worst case verification outcome among all 
    // verification providers which are supported
    bool is_any_verification_run_passed = false;
    for(auto &m : results_.back().verification_map) {
      if(m.second == Disposition::kFailed || m.second == Disposition::kIncorrect) {
        results_.back().disposition = m.second;
        return true;
      }
      if(!is_any_verification_run_passed && m.second == Disposition::kPassed) {
        is_any_verification_run_passed = true;
      }
    }

    if(is_any_verification_run_passed) {
      results_.back().disposition = Disposition::kPassed;
    }
  }

  // Return true means continue profiling
  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GemmOperationProfiler::verify_with_cublas_(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {


#if CUTLASS_ENABLE_CUBLAS

  library::GemmDescription const &gemm_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  //
  // Construct cuBLAS operators
  //
    
  CublasCreate handle;
  hipblasStatus_t status = handle.get_cublas_create_status();

  if (status != HIPBLAS_STATUS_SUCCESS) {

    results_.back().verification_map[library::Provider::kCUBLAS] = get_cutlass_disposition(status);
    return true;
  }

  std::vector<hipblasGemmAlgo_t> algorithms;

  detail::select_cublas_algorithms(
    algorithms, 
    options, 
    gemm_desc);

  if (algorithms.empty()) {
    // no algorithm selected
    return true;
  }

  //
  // Initialize state
  //

  try {

    //
    // Construct dispatcher to hipblasGemmEx()
    //

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
    gemm_workspace_.arguments.C = gemm_workspace_.Reference->data();
    gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.Reference->batch_stride();
    gemm_workspace_.arguments.D = gemm_workspace_.Reference->data();
    gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Reference->batch_stride();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    detail::cublasGemmExDispatcher gemm_op( 
      gemm_desc, 
      gemm_workspace_.configuration,
      gemm_workspace_.arguments,
      algorithms.front()
    );

    if (gemm_op.status != Status::kSuccess) {
      results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotRun;
      return true;
    }

    results_.back().status = Status::kSuccess;

    status = gemm_op(handle);

    // Handle errors
    if (status != HIPBLAS_STATUS_SUCCESS) {

      results_.back().verification_map[library::Provider::kCUBLAS] = get_cutlass_disposition(status);
      return true;
    }

    //
    // Verify results
    //

    results_.back().verification_map[library::Provider::kCUBLAS] = compare_tensors(
      options,
      *gemm_workspace_.Computed,
      *gemm_workspace_.Reference,
      gemm_workspace_.Computed->batch_stride()
    );

    // Save workspace if incorrect
    if (options.verification.save_workspace == SaveWorkspace::kIncorrect && 
      results_.back().verification_map[library::Provider::kCUBLAS] == Disposition::kIncorrect) {

      save_workspace(
        device_context,
        options,
        gemm_desc,
        library::Provider::kCUTLASS,
        library::Provider::kCUBLAS);
    }
  }
  catch (...) {
    results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kFailed;
  }

#endif

  // Return true means continue profiling
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against host and device references
bool GemmOperationProfiler::verify_with_reference_(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::GemmDescription const &gemm_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  //
  // Initialize state
  //

  library::Provider references[] = {
    library::Provider::kReferenceDevice,
    library::Provider::kReferenceHost
  };

  for (auto provider : references) {

    // Skip providers that are not enabled
    if (!options.verification.provider_enabled(provider)) {
      continue;
    }

    void *ptr_A = gemm_workspace_.A->data();
    void *ptr_B = gemm_workspace_.B->data();
    void *ptr_C = gemm_workspace_.C->data();
    void *ptr_D = gemm_workspace_.Reference->data();

    // To support the host-side reference, conditionally allocate and
    // copy tensors to host memory.
    std::vector<uint8_t> host_data_A;
    std::vector<uint8_t> host_data_B;
    std::vector<uint8_t> host_data_C;
    std::vector<uint8_t> host_data_D;

    if (provider == library::Provider::kReferenceHost) {

      host_data_A.resize(gemm_workspace_.A->bytes());
      ptr_A = host_data_A.data();
      gemm_workspace_.A->copy_to_host(ptr_A);

      host_data_B.resize(gemm_workspace_.B->bytes());
      ptr_B = host_data_B.data();
      gemm_workspace_.B->copy_to_host(ptr_B);

      host_data_C.resize(gemm_workspace_.C->bytes());
      ptr_C = host_data_C.data();
      gemm_workspace_.C->copy_to_host(ptr_C);

      host_data_D.resize(gemm_workspace_.Reference->bytes());
      ptr_D = host_data_D.data();
    }

    //
    // Launch
    //

    library::Handle handle;

    handle.set_provider(provider);

    Status status = handle.gemm_universal(
      problem_.mode,
      gemm_workspace_.configuration.problem_size.m(),
      gemm_workspace_.configuration.problem_size.n(),
      gemm_workspace_.configuration.problem_size.k(),
      gemm_desc.tile_description.math_instruction.element_accumulator,
      gemm_desc.element_epilogue,

      problem_.alpha.data(),

      gemm_desc.A.element,
      gemm_desc.A.layout,
      gemm_desc.transform_A,
      ptr_A,
      int(gemm_workspace_.configuration.lda),

      gemm_desc.B.element,
      gemm_desc.B.layout,
      gemm_desc.transform_B,
      ptr_B,
      int(gemm_workspace_.configuration.ldb),

      problem_.beta.data(),

      gemm_desc.C.element,
      ptr_C,
      int(gemm_workspace_.configuration.ldc),

      ptr_D,
      int(gemm_workspace_.configuration.ldd),

      gemm_workspace_.configuration.batch_count,
      gemm_workspace_.A->batch_stride(),
      gemm_workspace_.B->batch_stride(),
      gemm_workspace_.C->batch_stride(),
      gemm_workspace_.Reference->batch_stride()
    );

    if (status != Status::kSuccess) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
      return true;
    }

    results_.back().status = status;

    if (provider == library::Provider::kReferenceHost) {
      gemm_workspace_.Reference->copy_from_host(ptr_D); 
    }

    //
    // Verify results
    //

    results_.back().verification_map[provider] = compare_tensors(
      options,
      *gemm_workspace_.Computed,
      *gemm_workspace_.Reference,
      gemm_workspace_.Computed->batch_stride()
    );

    // Save workspace if incorrect
    if (options.verification.save_workspace == SaveWorkspace::kIncorrect && 
      results_.back().verification_map[provider] == Disposition::kIncorrect) {

      save_workspace(
        device_context,
        options,
        gemm_desc,
        library::Provider::kCUTLASS,
        provider);
    }
  }

  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool GemmOperationProfiler::profile(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.C = gemm_workspace_.C->data();
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;
    gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
    gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
    gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.C->batch_stride();
    gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Computed->batch_stride();

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_.arguments.D                       = gemm_workspace_.device_workspace.data();
      gemm_workspace_.arguments.alpha                   = problem_.alpha_one.data();
      gemm_workspace_.arguments.beta                    = problem_.beta_zero.data();

      gemm_workspace_.reduction_arguments.workspace     = gemm_workspace_.device_workspace.data();
      gemm_workspace_.reduction_arguments.source        = gemm_workspace_.C->data();
      gemm_workspace_.reduction_arguments.destination   = gemm_workspace_.Computed->data();
      gemm_workspace_.reduction_arguments.alpha         = problem_.alpha.data();
      gemm_workspace_.reduction_arguments.beta          = problem_.beta.data();
      gemm_workspace_.reduction_arguments.pointer_mode  = library::ScalarPointerMode::kHost;
    }

    results_.back().status = profile_cutlass_(
      results_.back().runtime,
      options,
      operation,
      &gemm_workspace_.arguments,
      gemm_workspace_.host_workspace.data(),
      gemm_workspace_.device_workspace.data()
    );
  }
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Method to profile a CUTLASS Operation
Status GemmOperationProfiler::profile_cutlass_(
  double &runtime,
  Options const &options,
  library::Operation const *operation,
  void *arguments,
  void *host_workspace,
  void *device_workspace) {

  GpuTimer timer;

  // initialize gemm underlying operation to handle parallel reduction
  library::Operation const * underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      return Status::kErrorNotSupported;
    }
  }

  //
  // Optional sleep to limit power consumption and thermals
  //

  sleep(options.profiling.sleep_duration);

  //
  // Warmup loop
  //

  Status status;

  for (int iteration = 0; iteration < options.profiling.warmup_iterations; ++iteration) {
    
    int problem_idx = (iteration % gemm_workspace_.problem_count) * problem_.batch_count;

    gemm_workspace_.arguments.A = gemm_workspace_.A->batch_data(problem_idx);
    gemm_workspace_.arguments.B = gemm_workspace_.B->batch_data(problem_idx);
    gemm_workspace_.arguments.C = gemm_workspace_.C->batch_data(problem_idx);
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->batch_data(problem_idx);

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_.arguments.D                     = gemm_workspace_.device_workspace.data();

      gemm_workspace_.reduction_arguments.workspace   = gemm_workspace_.device_workspace.data();
      gemm_workspace_.reduction_arguments.source      = gemm_workspace_.C->batch_data(problem_idx);
      gemm_workspace_.reduction_arguments.destination = gemm_workspace_.Computed->batch_data(problem_idx);
    }

    // Execute the CUTLASS operation
    status = underlying_operation->run(
      &gemm_workspace_.arguments,
      host_workspace,
      device_workspace);

    if (status != Status::kSuccess) {
      return status;
    }

    // Run parallel reduction kernel for parallel split_k_mode
    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      status = reduction_op_->run(
        &gemm_workspace_.reduction_arguments,
        gemm_workspace_.reduction_host_workspace.data(),
        nullptr);

      if (status != Status::kSuccess) {
        return status;
      }
    }
  }

  //
  // Initialize GPU timer
  //

  timer.start();

  //
  // Profiling loop
  //

  int Iterations = options.profiling.iterations;

  int iteration = 0;
  for (; iteration < Iterations; ++iteration) {
    
    // Iterate over copies of the problem in memory
    int workspace_idx = options.profiling.warmup_iterations + iteration;
    int problem_idx = (workspace_idx % gemm_workspace_.problem_count) * problem_.batch_count;

    gemm_workspace_.arguments.A = gemm_workspace_.A->batch_data(problem_idx);
    gemm_workspace_.arguments.B = gemm_workspace_.B->batch_data(problem_idx);
    gemm_workspace_.arguments.C = gemm_workspace_.C->batch_data(problem_idx);
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->batch_data(problem_idx);

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_.arguments.D                     = gemm_workspace_.device_workspace.data();

      gemm_workspace_.reduction_arguments.workspace   = gemm_workspace_.device_workspace.data();
      gemm_workspace_.reduction_arguments.source      = gemm_workspace_.C->batch_data(problem_idx);
      gemm_workspace_.reduction_arguments.destination = gemm_workspace_.Computed->batch_data(problem_idx);
    }

    status = underlying_operation->run(
      arguments,
      host_workspace,
      device_workspace);

    if (status != Status::kSuccess) {
      return status;
    }

    // Run parallel reduction kernel for parallel split_k_mode
    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      status = reduction_op_->run(
        &gemm_workspace_.reduction_arguments,
        gemm_workspace_.reduction_host_workspace.data(),
        nullptr);

      if (status != Status::kSuccess) {
        return status;
      }
    }
  }

  //
  // Wait for completion
  //

  timer.stop_and_wait();

  //
  // Update performance result
  //

  runtime = timer.duration(iteration);

  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
