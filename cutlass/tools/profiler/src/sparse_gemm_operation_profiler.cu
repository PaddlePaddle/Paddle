#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment

*/

#include <iostream>
#include <stdexcept>
#include <iomanip>
#include <ios>

#include "cublas_helpers.h"
#include "sparse_gemm_operation_profiler.h"
#include "gpu_timer.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {


/////////////////////////////////////////////////////////////////////////////////////////////////

/// Ctor
SparseGemmOperationProfiler::SparseGemmOperationProfiler(Options const &options): 
  OperationProfiler(
    options,
    library::OperationKind::kSparseGemm,
    {
  	  {ArgumentTypeID::kEnumerated, {"gemm_kind"}, "Variant of GEMM (e.g. gemm, planar complex, batched, ...)"},
  	  {ArgumentTypeID::kInteger, {"m", "problem-size::m"}, "M dimension of the GEMM problem space"},
    	{ArgumentTypeID::kInteger, {"n", "problem-size::n"}, "N dimension of the GEMM problem space"},
	    {ArgumentTypeID::kInteger, {"k", "problem-size::k"}, "K dimension of the GEMM problem space"},
    	{ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
	    {ArgumentTypeID::kTensor, {"B"}, "Tensor storing the B operand"},
  	  {ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
  	  {ArgumentTypeID::kTensor, {"E"}, "Tensor storing the E operand"},
  	  {ArgumentTypeID::kScalar, {"alpha", "epilogue::alpha"}, "Epilogue scalar alpha"},
    	{ArgumentTypeID::kScalar, {"beta", "epilogue::beta"}, "Epilogue scalar beta"},
	    {ArgumentTypeID::kInteger, {"split_k_slices"}, "Number of partitions of K dimension"},
    	{ArgumentTypeID::kInteger, {"batch_count"}, "Number of GEMMs computed in one batch"},
    }
  ) {

  description_ = "      Structured sparse GEMM. D = alpha * A*B + beta * C";
}

/// Destructor
SparseGemmOperationProfiler::~SparseGemmOperationProfiler() {

}

/// Prints usage statement for the math function
void SparseGemmOperationProfiler::print_usage(std::ostream &out) const {
  out << "Sparse GEMM" << "\n\n";

  OperationProfiler::print_usage(out);
}

/// Prints examples
void SparseGemmOperationProfiler::print_examples(std::ostream &out) const {

  out << "\nExamples:\n\n"
    << "Profile a particular problem size:\n"
    << "  $ cutlass_profiler --operation=SparseGemm --m=1024 --n=1024 --k=128\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=SparseGemm --m=1024:4096:256 --n=1024:4096:256 --k=128:8192:128 --beta=0,1,2.5\n\n"

    << "Schmoo over accumulator types:\n"
    << "  $ cutlass_profiler --operation=SparseGemm --accumulator-type=f16,f32\n\n"

    << "Run when A is f16 with column-major and B is any datatype with row-major (For column major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=SparseGemm --A=f16:column --B=*:row\n\n"

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=SparseGemm --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=SparseGemm --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=SparseGemm --dist=sequential,start:0,delta:1\n\n"

    << "Run a kernel with cta tile size of 256x128x32 and save workspace if results are incorrect (note that --cta-tile::k=32 is default cta-tile size):\n"
    << " $ cutlass_profiler --operation=SparseGemm --cta_m=256 --cta_n=128  --cta_k=32 --save-workspace=incorrect\n\n"
    
    << "Test your changes to gemm kernels with a quick functional test and save results in functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=SparseGemm \\ \n"
    << "   --m=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --n=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --k=8,16,32,64,128,256,288,384,504,512,520 \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

/////////////////////////////////////////////////////////////////////////////////////////////////

Status SparseGemmOperationProfiler::SparseGemmProblem::parse(
  library::SparseGemmDescription const &operation_desc,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {
  
  if (!arg_as_int(this->m, "m", problem_space, problem)) {
    // default value
    this->m = 1024;
  }

  if (!arg_as_int(this->n, "n", problem_space, problem)) {
    // default value
    this->n = 1024;
  }
  
  if (!arg_as_int(this->k, "k", problem_space, problem)) {
    // default value
    this->k = 1024;
  }
  
  if (!arg_as_int(this->split_k_slices, "split_k_slices", problem_space, problem)) {
    // default value
    this->split_k_slices = 1;
  }
  
  if (!arg_as_int(this->batch_count, "batch_count", problem_space, problem)) {
    // default value
    this->batch_count = 1;
  }

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.B, "B", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.E, "E", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
    this->alpha, 
    operation_desc.element_epilogue, 
    "alpha", 
    problem_space, 
    problem)) {

    if (!cast_from_double(this->alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }
  
  if (!arg_as_scalar(
    this->beta, 
    operation_desc.element_epilogue, 
    "beta", 
    problem_space, 
    problem)) {
    
    if (!cast_from_double(this->beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }

  this->elements_per_128b =
      128 / library::sizeof_bits(operation_desc.A.element);

  this->lda = DeviceAllocation::get_packed_layout(
                  operation_desc.A.layout,
                  {int(this->m), int(this->k) / int(this->sparse)})
                  .front();

  this->ldb = DeviceAllocation::get_packed_layout(
    operation_desc.B.layout, {int(this->k), int(this->n)}).front();

  this->ldc = DeviceAllocation::get_packed_layout(
    operation_desc.C.layout, {int(this->m), int(this->n)}).front();

  this->lde =
      DeviceAllocation::get_packed_layout(
          operation_desc.E.layout,
          {int(this->m), int(this->k / this->sparse / this->elements_per_128b)})
          .front();

  return Status::kSuccess;
}

/// Initializes a performance result
void SparseGemmOperationProfiler::SparseGemmProblem::initialize_result(
  PerformanceResult &result,
  library::SparseGemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.arguments.resize(problem_space.rank());

  set_argument(result, "gemm_kind", problem_space, library::to_string(operation_desc.gemm_kind));

  set_argument(result, "A", problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" + library::to_string(operation_desc.A.layout));

  set_argument(result, "B", problem_space,
    std::string(library::to_string(operation_desc.B.element)) + ":" + library::to_string(operation_desc.B.layout));

  set_argument(result, "C", problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" + library::to_string(operation_desc.C.layout));

  set_argument(result, "E", problem_space,
    std::string(library::to_string(operation_desc.E.element)) + ":" + library::to_string(operation_desc.E.layout));

  set_argument(result, "m", problem_space, m);
  set_argument(result, "n", problem_space, n);
  set_argument(result, "k", problem_space, k);

  set_argument(result, "split_k_slices", problem_space, split_k_slices);
  set_argument(result, "batch_count", problem_space, batch_count);

  set_argument(result, "alpha", problem_space,
    library::lexical_cast(alpha, operation_desc.element_epilogue));

  set_argument(result, "beta", problem_space,
    library::lexical_cast(beta, operation_desc.element_epilogue));
}

/// Extracts the problem dimensions
Status SparseGemmOperationProfiler::initialize_configuration(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::SparseGemmDescription const &operation_desc = 
    static_cast<library::SparseGemmDescription const &>(operation->description());

  if (operation_desc.gemm_kind != library::GemmKind::kSparse) {
    return Status::kErrorInvalidProblem;
  }

  Status status = problem_.parse(operation_desc, problem_space, problem);

  if (status != Status::kSuccess) {
    return status;
  }

  gemm_workspace_.configuration.problem_size.m() = int(problem_.m);
  gemm_workspace_.configuration.problem_size.n() = int(problem_.n);
  gemm_workspace_.configuration.problem_size.k() = int(problem_.k);
  gemm_workspace_.configuration.lda = problem_.lda;
  gemm_workspace_.configuration.ldb = problem_.ldb;
  gemm_workspace_.configuration.ldc = problem_.ldc;
  gemm_workspace_.configuration.ldd = problem_.ldc;
  gemm_workspace_.configuration.lde = problem_.lde;

  gemm_workspace_.arguments.A = nullptr;
  gemm_workspace_.arguments.B = nullptr;
  gemm_workspace_.arguments.C = nullptr;
  gemm_workspace_.arguments.D = nullptr;
  gemm_workspace_.arguments.E = nullptr;
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  initialize_result_(this->model_result_, options, operation_desc, problem_space);
  
  return operation->can_implement(&gemm_workspace_.configuration, &gemm_workspace_.arguments);
}

/// Initializes the performance result
void SparseGemmOperationProfiler::initialize_result_(
  PerformanceResult &result,
  Options const &options,  
  library::SparseGemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.provider = library::Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;

  problem_.initialize_result(result, operation_desc, problem_space);
  
  OperationProfiler::initialize_result_(result, operation_desc, problem_space);

  // Input bytes read and Output bytes written for the gemm problem
  result.bytes =
      int64_t(library::sizeof_bits(operation_desc.A.element) * problem_.m / 8) *
          problem_.k / problem_.sparse +
      int64_t(library::sizeof_bits(operation_desc.B.element) * problem_.n / 8) *
          problem_.k +
      int64_t(library::sizeof_bits(operation_desc.C.element) * problem_.m / 8) *
          problem_.n +
      int64_t(library::sizeof_bits(operation_desc.E.element) * problem_.m / 8) *
          problem_.k / problem_.sparse / problem_.elements_per_128b;

  // Set is_beta_zero true if beta is zero
  bool is_beta_zero = std::all_of(problem_.beta.begin(), problem_.beta.end(), [](uint8_t i) { return i==0; });

  // Output bytes read for the gemm problem for non-zero beta values
  if (!is_beta_zero) {
    result.bytes += int64_t(library::sizeof_bits(operation_desc.C.element) * problem_.m / 8) * problem_.n;
  }

  result.flops = 2 * (problem_.m * problem_.n * problem_.k + problem_.m * problem_.n);
  result.runtime = 0;

}

/// Initializes workspace
Status SparseGemmOperationProfiler::initialize_workspace(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {
  
  library::SparseGemmDescription const &operation_desc = 
    static_cast<library::SparseGemmDescription const &>(operation->description());

  if (options.execution_mode != ExecutionMode::kDryRun) {

    gemm_workspace_.A = device_context.allocate_tensor(
      options,
      "A",
      operation_desc.A.element,
      operation_desc.A.layout,
      {int(problem_.m), int(problem_.k) / int(problem_.sparse)},
      {int(problem_.lda)}
    );

    gemm_workspace_.B = device_context.allocate_tensor(
      options,
      "B",
      operation_desc.B.element,
      operation_desc.B.layout,
      {int(problem_.k), int(problem_.n)},
      {int(problem_.ldb)}
    );

    gemm_workspace_.C = device_context.allocate_tensor(
      options,
      "C",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)}
    );

    gemm_workspace_.Computed = device_context.allocate_tensor(
      "D",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)}
    );

    gemm_workspace_.E = device_context.allocate_sparsemeta_tensor(
      options,
      "E",
      operation_desc.E.element,
      operation_desc.E.layout,
      operation_desc.A.element,
      {int(problem_.m), int(problem_.k) / int(problem_.sparse) / int(problem_.elements_per_128b)},
      {int(problem_.lde)}
    );

    gemm_workspace_.Reference = device_context.allocate_tensor(
      "Reference",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)}
    );

    gemm_workspace_.Reference->copy_from_device(gemm_workspace_.C->data());
  }

  //
  // Initialize the CUTLASS operation
  //

  Status status = Status::kSuccess;

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    if (options.execution_mode != ExecutionMode::kDryRun) {

      uint64_t workspace_size = operation->get_host_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.host_workspace.resize(workspace_size, 0);

      workspace_size = operation->get_device_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

      status = operation->initialize(
        &gemm_workspace_.configuration,
        gemm_workspace_.host_workspace.data(),
        gemm_workspace_.device_workspace.data());
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //

    results_.push_back(model_result_);
    results_.back().provider = library::Provider::kCUTLASS;
    results_.back().op_kind = library::OperationKind::kSparseGemm;
    results_.back().disposition = Disposition::kNotRun;

    for(auto &verification_provider : options.verification.providers) {
      results_.back().verification_map[verification_provider] = Disposition::kNotRun;
    }
  }

  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool SparseGemmOperationProfiler::verify_cutlass(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (!options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  // Initialize structure containing GEMM arguments
  gemm_workspace_.arguments.A = gemm_workspace_.A->data();
  gemm_workspace_.arguments.B = gemm_workspace_.B->data();
  gemm_workspace_.arguments.C = gemm_workspace_.C->data();
  gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
  gemm_workspace_.arguments.E = gemm_workspace_.E->data();
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  //
  // Run the CUTLASS operation
  //

  results_.back().status = operation->run(
    &gemm_workspace_.arguments, 
    gemm_workspace_.host_workspace.data(),
    gemm_workspace_.device_workspace.data());

  if (results_.back().status != Status::kSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // CUTLASS op ran the but not yet verified against any verification provider
  results_.back().disposition = Disposition::kNotVerified;

  //
  // Run verification providers
  //

  if (options.verification.enabled) {

    // Update disposition to worst case verification outcome among all 
    // verification providers which are supported
    bool is_any_verification_run_passed = false;

    for(auto &m : results_.back().verification_map) {
      if(m.second == Disposition::kFailed || m.second == Disposition::kIncorrect) {
        results_.back().disposition = m.second;
        return true;
      }
      if(!is_any_verification_run_passed && m.second == Disposition::kPassed) {
        is_any_verification_run_passed = true;
      }
    }

    if(is_any_verification_run_passed) {
      results_.back().disposition = Disposition::kPassed;
    }
  }

  // Return true means continue profiling
  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool SparseGemmOperationProfiler::profile(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.C = gemm_workspace_.C->data();
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
    gemm_workspace_.arguments.E = gemm_workspace_.E->data();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    results_.back().status = profile_cutlass_(
      results_.back().runtime,
      options,
      operation,
      &gemm_workspace_.arguments,
      gemm_workspace_.host_workspace.data(),
      gemm_workspace_.device_workspace.data()
    );
  }
  
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
