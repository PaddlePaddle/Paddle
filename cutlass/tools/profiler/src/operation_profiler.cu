/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Defines a math function
*/

#include <algorithm>
#include <stdexcept>
#include <iomanip>
#include <cstring>
#include <fstream>
#include <sstream>

#ifdef __unix__
#include <unistd.h>
#elif defined(_WIN32) || defined(WIN32)
#include <windows.h>
#else
// sleep not supported
#endif

#include "options.h"
#include "operation_profiler.h"
#include "gpu_timer.h"

///////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {

///////////////////////////////////////////////////////////////////////////////////////////////////

OperationProfiler::OperationProfiler(): kind_(library::OperationKind::kInvalid) { }

/// Ctor
OperationProfiler::OperationProfiler(
  Options const &options,
  library::OperationKind kind,
  ArgumentDescriptionVector const &arguments,
  ProviderVector const & verification_providers
): 
  kind_(kind), arguments_(arguments) {

  ArgumentDescriptionVector tile_description_arguments{
    {ArgumentTypeID::kEnumerated, {"op_class", "opcode-class"}, "Class of math instruction (simt, tensorop, wmmatensorop, wmma)"},
    {ArgumentTypeID::kEnumerated, {"accum", "accumulator-type"}, "Math instruction accumulator data type"},
    {ArgumentTypeID::kInteger, {"cta_m", "threadblock-shape::m"}, "Threadblock shape in the M dimension"},
    {ArgumentTypeID::kInteger, {"cta_n", "threadblock-shape::n"}, "Threadblock shape in the N dimension"},
    {ArgumentTypeID::kInteger, {"cta_k", "threadblock-shape::k"}, "Threadblock shape in the K dimension"},
    {ArgumentTypeID::kInteger, {"stages", "threadblock-stages"}, "Number of stages of threadblock-scoped matrix multiply"},
    {ArgumentTypeID::kInteger, {"warps_m", "warp-count::m"}, "Number of warps within threadblock along the M dimension"},
    {ArgumentTypeID::kInteger, {"warps_n", "warp-count::n"}, "Number of warps within threadblock along the N dimension"},
    {ArgumentTypeID::kInteger, {"warps_k", "warp-count::k"}, "Number of warps within threadblock along the K dimension"},
    {ArgumentTypeID::kInteger, {"inst_m", "instruction-shape::m"}, "Math instruction shape in the M dimension"},
    {ArgumentTypeID::kInteger, {"inst_n", "instruction-shape::n"}, "Math instruction shape in the N dimension"},
    {ArgumentTypeID::kInteger, {"inst_k", "instruction-shape::k"}, "Math instruction shape in the K dimension"},
    {ArgumentTypeID::kInteger, {"min_cc", "minimum-compute-capability"}, "Minimum device compute capability"},
    {ArgumentTypeID::kInteger, {"max_cc", "maximum-compute-capability"}, "Maximum device compute capability"}
  };

  arguments_.insert(arguments_.end(), tile_description_arguments.begin(), tile_description_arguments.end());

  for (auto provider : verification_providers) {
    if (std::find(
      options.verification.providers.begin(), 
      options.verification.providers.end(), 
      provider) != options.verification.providers.end()) {

      verification_providers_.push_back(provider);
    }
  }
}

/// Destructor
OperationProfiler::~OperationProfiler() {

}

/// Gets the schema description
std::string const & OperationProfiler::description() const {
  return description_;
}

/// Prints usage statement for the math function
void OperationProfiler::print_usage(std::ostream &out) const {
  for (auto const & desc : arguments_) {

    size_t const kAliasStart = 10;

    size_t columns = 0;
    
    std::string type_str = to_string(desc.type);
    columns += type_str.size();

    out << "  [" << type_str << "]";

    if (columns < kAliasStart) {
      out << std::string(kAliasStart - columns, ' ');  
    }

    columns = 0;

    int j = 0;
    for (auto const & alias : desc.aliases) {
      columns += alias.size() + (j ? 1 : 0) + 2;

      out << (j++ ? "," : "") << "--" << alias;
    }

    size_t const kTotalColumns = 50;

    if (columns < kTotalColumns) {
      out << std::string(kTotalColumns - columns, ' ');
    }

    out << desc.description << "\n";
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns true if the current operation description satisfies the problem space
bool OperationProfiler::satisfies(
  library::OperationDescription const &op_desc,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::OpcodeClassID opcode_class;
  if (arg_as_OpcodeClassID(opcode_class, "op_class", problem_space, problem)) {
    if (opcode_class != op_desc.tile_description.math_instruction.opcode_class) {
      return false;
    }
  }
  
  int64_t int_value;

  if (arg_as_int(int_value, "inst_m", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.math_instruction.instruction_shape.m()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "inst_n", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.math_instruction.instruction_shape.n()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "inst_k", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.math_instruction.instruction_shape.k()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "cta_m", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.threadblock_shape.m()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "cta_n", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.threadblock_shape.n()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "cta_k", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.threadblock_shape.k()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "stages", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.threadblock_stages) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "warps_m", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.warp_count.m()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "warps_n", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.warp_count.n()) != int_value) {
      return false;
    }
  }

  if (arg_as_int(int_value, "warps_k", problem_space, problem)) {
    if (int64_t(op_desc.tile_description.warp_count.k()) != int_value) {
      return false;
    }
  }

  library::NumericTypeID numeric_type;
  if (arg_as_NumericTypeID(numeric_type, "accum", problem_space, problem)) {
    if (numeric_type != op_desc.tile_description.math_instruction.element_accumulator) {
      return false;
    }
  }

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Entry point to profile all operations in the manifest
int OperationProfiler::profile_all(
  Options const &options, 
  library::Manifest const &manifest, 
  DeviceContext &device_context) {
  
  ProblemSpace problem_space(arguments_, options.cmdline);

  // 1. Construct performance report
  PerformanceReport report(options, problem_space.argument_names(), kind_);

  // 2. For each problem in problem space
  ProblemSpace::Iterator problem_it = problem_space.begin();
  ProblemSpace::Iterator problem_end = problem_space.end();

  bool continue_profiling = true, internal_error = false;

  // For each problem in problem space
  for (; continue_profiling && problem_it != problem_end; ++problem_it) {

    ProblemSpace::Problem problem = problem_it.at();

    report.next_problem();

    // For each operation in manifest
    for (auto const & operation_ptr : manifest) {

      library::Operation const *operation = operation_ptr.get();

      auto min_cc = operation->description().tile_description.minimum_compute_capability;
      auto max_cc = operation->description().tile_description.maximum_compute_capability;

      // Clear named allocations
      device_context.free();

      // Execute compatible cutlass operations if they satisfy the current device's compute capability
      if (operation->description().kind == kind_ &&
        operation->description().provider == library::Provider::kCUTLASS &&
        options.device.compute_capability() >= min_cc &&
          options.device.compute_capability() <= max_cc) {

        std::string operation_name(operation->description().name);

        // Filter kernels by name
        bool filtered_by_name = options.operation_names.empty();
        if (!filtered_by_name) {
          
          for (auto const & op_name : options.operation_names) {
            if (find_string_matches_(op_name, operation_name)) {
              filtered_by_name = true;
              break;
            }
          } 
        }

        for (auto const & op_name : options.excluded_operation_names) {
          if (find_string_matches_(op_name, operation_name)) {
            filtered_by_name = false;
            break;
          }
        }

        if (!filtered_by_name || !satisfies(operation->description(), problem_space, problem)) {
          continue;
        }
      
        // A. Initialize configuration
        Status status = this->initialize_configuration(
          options,
          report,
          device_context,
          operation,
          problem_space,
          problem);

        if (status == Status::kErrorInternal) {
          
          // If there was an internal error, consume the CUDA error and move to the next operation.
          (void)hipGetLastError();
          
          report.append_results(results_);
          continue;
        }
        else if (status != Status::kSuccess) {
          // If the workspace could not be initialized for any other reason, continue to
          // the next operation.
          continue;
        }

        if (continue_profiling) {

          status = this->initialize_workspace(
            options,
            report,
            device_context,
            operation,
            problem_space,
            problem);

          if (status == Status::kErrorInternal) {

            // If there was an internal error, consume the CUDA error and move to the next operation.
            (void)hipGetLastError();

            report.append_results(results_);
            continue;
          }
          else if (status != Status::kSuccess) {
            // If the workspace could not be initialized for any other reason, continue to
            // the next operation.
            continue;
          }
        }

        //
        // Profile CUTLASS if it is enabled
        //

        // B. Verify CUTLASS
         
        if (continue_profiling && options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

          continue_profiling = this->verify_cutlass(
            options,
            report, 
            device_context, 
            operation, 
            problem_space,
            problem);
        }

        if (options.execution_mode == ExecutionMode::kDryRun) {
          report.append_results(results_);
          results_.clear();
          continue;
        }

        //
        // C. Optionally save workspace
        //

        if (options.verification.save_workspace == SaveWorkspace::kAlways) {
          save_workspace(
            device_context,
            options,
            operation->description(),
            library::Provider::kCUTLASS);
        }

        //
        // D. Profile
        //

        if (continue_profiling && options.profiling.enabled) {

          continue_profiling = this->profile(
            options, 
            report, 
            device_context, 
            operation, 
            problem_space,
            problem);
        }

        report.append_results(results_);
        results_.clear();
      }

      if (!continue_profiling) {
        break;
      }
    } 
  }

  return internal_error ? 1 : 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Sleep for a given duration in ms
void OperationProfiler::sleep(int sleep_duration) {
  if (sleep_duration) {
    #ifdef __unix__
    usleep(sleep_duration * 1000);
    #elif defined(_WIN32) || defined(WIN32)
    SleepEx(sleep_duration, false);
    #else
    // sleep not supported
    #endif 
  }
}


/// Compares tensors for equality
Disposition OperationProfiler::compare_tensors(
  Options const &options,
  DeviceAllocation &experimental,
  DeviceAllocation &reference,
  int64_t count) {

  if (experimental.type() != reference.type()) {
    return Disposition::kIncorrect;
  }

  bool passed = false;

  if (count == 0) {
    count = reference.capacity();
  }

  if (options.verification.epsilon == 0) {

    // bit-level equality
    passed = DeviceAllocation::block_compare_equal(
      experimental.type(), 
      experimental.data(),
      reference.data(),
      count);
  }
  else {

    // relative error function
    passed = DeviceAllocation::block_compare_relatively_equal(
      experimental.type(), 
      experimental.data(),
      reference.data(),
      count,
      options.verification.epsilon,
      options.verification.nonzero_floor);
  }

  return passed ? Disposition::kPassed : Disposition::kIncorrect;
}

/// Saves the workspace
void OperationProfiler::save_workspace(
  DeviceContext &device_context,
  Options const &options,
  library::OperationDescription const &desc,
  library::Provider provider,
  library::Provider verification_provider) {

  for (auto const & named_allocation : device_context) {

    DeviceAllocation *allocation = named_allocation.second;
    
    std::stringstream filename;

    filename << desc.name << "_" << library::to_string(provider) << "_";

    if (verification_provider != library::Provider::kInvalid) {
      filename << "verified_by_" << library::to_string(verification_provider) << "_";
    }

    filename << named_allocation.first + ".mat";

    std::ofstream out(filename.str());

    allocation->write_tensor_csv(out);
    out << "\n";

    if (options.report.verbose) {
      std::cout << "wrote '" << filename.str() << "'" << std::endl;
    }
  } 
}


///////////////////////////////////////////////////////////////////////////////////////////////////

/// Method to profile a CUTLASS Operation
Status OperationProfiler::profile_cutlass_(
  double &runtime,
  Options const &options,
  library::Operation const *operation,
  void *arguments,
  void *host_workspace,
  void *device_workspace) {

  GpuTimer timer;

  //
  // Optional sleep to limit power consumption and thermals
  //

  sleep(options.profiling.sleep_duration);

  //
  // Warmup loop
  //

  Status status;

  for (int iteration = 0; iteration < options.profiling.warmup_iterations; ++iteration) {

    status = operation->run(
      arguments,
      host_workspace,
      device_workspace);

    if (status != Status::kSuccess) {
      return status;
    }
  }
  
  //
  // Initialize GPU timer
  //

  timer.start();

  //
  // Profiling loop
  //

  int Iterations = options.profiling.iterations;

  int iteration = 0;
  for (; iteration < Iterations; ++iteration) {
    
    status = operation->run(
      arguments,
      host_workspace,
      device_workspace);

    if (status != Status::kSuccess) {
      return status;
    }
  }

  //
  // Wait for completion
  //

  timer.stop_and_wait();

  //
  // Update performance result
  //
  
  runtime = timer.duration(iteration);

  return status;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Sets operation description 
void OperationProfiler::initialize_result_(
  PerformanceResult &result,
  library::OperationDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  set_argument(result, "op_class", problem_space,
    library::to_string(operation_desc.tile_description.math_instruction.opcode_class));

  set_argument(result, "accum", problem_space,
    library::to_string(operation_desc.tile_description.math_instruction.element_accumulator));

  set_argument(result, "cta_m", problem_space, operation_desc.tile_description.threadblock_shape.m());
  set_argument(result, "cta_n", problem_space, operation_desc.tile_description.threadblock_shape.n());
  set_argument(result, "cta_k", problem_space, operation_desc.tile_description.threadblock_shape.k());
  set_argument(result, "stages", problem_space, operation_desc.tile_description.threadblock_stages);
  set_argument(result, "warps_m", problem_space, operation_desc.tile_description.warp_count.m());
  set_argument(result, "warps_n", problem_space, operation_desc.tile_description.warp_count.n());
  set_argument(result, "warps_k", problem_space, operation_desc.tile_description.warp_count.k());
  set_argument(result, "inst_m", problem_space, operation_desc.tile_description.math_instruction.instruction_shape.m());
  set_argument(result, "inst_n", problem_space, operation_desc.tile_description.math_instruction.instruction_shape.n());
  set_argument(result, "inst_k", problem_space, operation_desc.tile_description.math_instruction.instruction_shape.k());
  set_argument(result, "min_cc", problem_space, operation_desc.tile_description.minimum_compute_capability);
  set_argument(result, "max_cc", problem_space, operation_desc.tile_description.maximum_compute_capability);
}

/// Helper
void OperationProfiler::set_argument(
  PerformanceResult &result,
  char const *name,
  ProblemSpace const &problem_space,
  std::string const &value) {

  result.arguments.at(problem_space.argument_index(name)) = make_pair(std::string(name), value);
}

void OperationProfiler::set_argument(  
  PerformanceResult &result,
  char const *name,
  ProblemSpace const &problem_space,
  int64_t value) {

  result.arguments.at(problem_space.argument_index(name)) = make_pair(std::string(name), library::lexical_cast(value));
}


/// finds string matches filter_string in operation_name
bool OperationProfiler::find_string_matches_(
  std::string const &filter_string, 
  std::string const &operation_name) {
  // Returns true if all substrings appear in the operation_name in order
  
  // Split filter_string of the format "gemm*f32*nt" to tokens ["gemm", "f32", "nt"]
  std::string item;  
  std::istringstream iss(filter_string);
  std::vector<std::string> filter_tokens;
  while (std::getline(iss, item, '*')) {
    filter_tokens.push_back(item);
  }

  // Search filter_tokens in operation_name in order
  size_t start = 0, idx = 0;
  for(auto & token : filter_tokens) {
    // Check if characters left to be parsed in operation_name
    if (start < operation_name.length()) {
      // Find token in operation_name[start:]
      idx = operation_name.substr(start).find(token);
      if (idx == std::string::npos) {
        return false;
      }
    }
    start += (idx + token.length()); 
  }

  // All tokens in filter_string found in operation_name
  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////////////////////////
