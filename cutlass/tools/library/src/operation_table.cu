#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*
  \file
  \brief Defines a data structure in which a set of functionally equivalent library::Operation
        instances may be queried.
*/

#include "cutlass/library/operation_table.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

/////////////////////////////////////////////////////////////////////////////////////////////////

void OperationTable::append(Manifest const &manifest) {

  // Insert operations into appropriate data structure
  for (auto const & operation : manifest) {

    OperationDescription const &desc = operation->description();

    // insert all gemm operation into operation table
    if (desc.kind == OperationKind::kGemm) {
      GemmDescription const &gemm_desc = static_cast<GemmDescription const &>(desc);
    

      GemmFunctionalKey functional_key(
        gemm_desc.provider,
        gemm_desc.gemm_kind,
        gemm_desc.tile_description.math_instruction.element_accumulator,
        gemm_desc.element_epilogue,
        gemm_desc.A.element,
        gemm_desc.A.layout,
        gemm_desc.transform_A,
        gemm_desc.B.element,
        gemm_desc.B.layout,
        gemm_desc.transform_B,
        gemm_desc.C.element
      );

      Operation const *op = operation.get();

      int cc = gemm_desc.tile_description.minimum_compute_capability;
        
      int alignment = std::max(std::max(
        gemm_desc.A.alignment, gemm_desc.B.alignment), gemm_desc.C.alignment);

      GemmPreferenceKey preference_key(cc, alignment);

      gemm_operations[functional_key][preference_key].push_back(op);
    }

    // insert all conv2d or conv3d operation into operation table
    if (desc.kind == OperationKind::kConv2d || desc.kind == OperationKind::kConv3d) {
      auto &conv_desc = static_cast<library::ConvDescription const &>(desc);

      ConvFunctionalKey functional_key(
        conv_desc.provider,
        conv_desc.conv_kind,
        conv_desc.A.element,
        conv_desc.A.layout,
        conv_desc.B.element,
        conv_desc.B.layout,
        conv_desc.C.element,
        conv_desc.C.layout,
        conv_desc.tile_description.math_instruction.element_accumulator, 
        conv_desc.element_epilogue
      );

      Operation const *op = operation.get();

      int cc = conv_desc.tile_description.minimum_compute_capability;

      ConvPreferenceKey preference_key(cc, conv_desc.iterator_algorithm);

      // insert conv operation to conv2d_operations or conv3d_operations map
      (desc.kind == OperationKind::kConv2d) ?
        conv2d_operations[functional_key][preference_key].push_back(op) : 
        conv3d_operations[functional_key][preference_key].push_back(op);
    }

    // insert all reduction operation into operation table
    if (desc.kind == OperationKind::kReduction) {
      auto &reduce_desc = static_cast<library::ReductionDescription const &>(desc);

      ReductionFunctionalKey functional_key(
        reduce_desc.provider,
        reduce_desc.element_workspace,
        reduce_desc.tile_description.math_instruction.element_accumulator,
        reduce_desc.element_output,
        reduce_desc.element_epilogue,
        library::MathOperationID::kAdd,
        library::EpilogueKind::kLinearCombination
      );

      Operation const *op = operation.get();

      reduction_operations[functional_key] = op;

    }

  }

}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace library
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////

