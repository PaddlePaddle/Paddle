#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief CUTLASS Library handle.
*/
#include <iostream> 
#include <stdexcept>
#include <cstdint>

#include "cutlass/library/handle.h"
#include "cutlass/library/singleton.h"
#include "cutlass/library/util.h"

namespace cutlass {
namespace library {

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Constructor
Handle::Handle(
  hipStream_t stream, 
  size_t workspace_size
):
  provider_(Provider::kCUTLASS), 
  stream_(stream), 
  workspace_(nullptr), 
  workspace_size_(0), 
  scalar_pointer_mode_(ScalarPointerMode::kHost), 
  last_operation_(nullptr) {

  int device_idx = -1;

  hipError_t error = hipGetDevice(&device_idx);
  if (error != hipSuccess) {
    throw std::runtime_error("hipGetDevice() failed");
  }

  error = hipGetDeviceProperties(&device_, device_idx);
  if (error != hipSuccess) {
    throw std::runtime_error("hipGetDeviceProperties() failed");
  }

  set_workspace_size(workspace_size);

  Singleton::get();
}

/// Destructor
Handle::~Handle() {
  if (workspace_) {

    if (workspace_) {
      hipFree(workspace_);
    }

    workspace_ = nullptr;
    workspace_size_ = 0;
  }
}

/// Move constructor
Handle::Handle(Handle && handle) {
  device_ = handle.device_;
  workspace_size_ = handle.workspace_size_;
  workspace_ = handle.workspace_;
  stream_ = handle.stream_;
  scalar_pointer_mode_ = handle.scalar_pointer_mode_;
  
  handle.workspace_ = nullptr;
  handle.workspace_size_ = 0;
}

/// Move assignment operator
Handle & Handle::operator=(Handle && handle) {

  provider_ = handle.provider_;
  device_ = handle.device_;
  workspace_size_ = handle.workspace_size_;
  workspace_ = handle.workspace_;
  stream_ = handle.stream_;
  scalar_pointer_mode_ = handle.scalar_pointer_mode_;

  handle.workspace_ = nullptr;
  handle.workspace_size_ = 0;

  return *this;
}

int Handle::compute_capability() const {
  return device_.major * 10 + device_.minor;
}

/// Sets the current CUDA stream
void Handle::set_stream(hipStream_t stream) {
  stream_ = stream;
}

/// Gets the current CUDA stream
hipStream_t Handle::get_stream() const {
  return stream_;
}

/// Gets the current provider
Provider Handle::get_provider() const {
  return provider_;
}

/// Sets the provider of operations
void Handle::set_provider(Provider provider) {
  provider_ = provider;
}

/// Gets the device workspace size
size_t Handle::get_workspace_size() const {
  return workspace_size_;
}

/// Gets a pointer to the device workspace allocation in Global Memory
void *Handle::get_workspace() const {
  return workspace_;
}

/// Sets the size of device workspace, invalidating previous calls to get_device_workspace()
void Handle::set_workspace_size(size_t bytes) {
  if (bytes != workspace_size_) {

    if (workspace_) {
      hipFree(workspace_);
    }
      
    workspace_ = nullptr;
    workspace_size_ = bytes;

    if (workspace_size_) {
  
      hipError_t error = hipMalloc((void **)&workspace_, workspace_size_);
  
      if (error != hipSuccess) {
        throw std::runtime_error("Failed to allocate workspace");
      }
    }
  }

  if (workspace_) {
    hipError_t error = hipMemset(workspace_, 0, workspace_size_);

    if (error != hipSuccess) {
      throw std::runtime_error("Failed to clear workspace");
    }
  }
}

/// Gets the scalar pointer mode
ScalarPointerMode Handle::get_scalar_pointer_mode() const {
  return scalar_pointer_mode_;
}

/// Sets the scalar pointer mode
void Handle::set_scalar_pointer_mode(ScalarPointerMode mode) {
  scalar_pointer_mode_ = mode;
}

/// Gets the last operation
Operation const *Handle::get_last_operation() const {
  return last_operation_;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns the maximum required alignment for each operator
static int maximum_alignment_requirement(GemmDescription const &desc) {
  return std::max(
    std::max(desc.A.alignment, desc.B.alignment), desc.C.alignment);
}

/// Returns the largest alignment (in units of elements) the problem satisfies, starting from a
/// given upper limit.
static int gemm_problem_alignment(
  int M,
  int N,
  int K,
  NumericTypeID element_A,
  void const *ptr_A,
  int64_t lda,
  int64_t batch_stride_A,
  NumericTypeID element_B,
  void const *ptr_B,
  int64_t ldb,
  int64_t batch_stride_B,
  NumericTypeID element_C,
  void const * ptr_C,
  int64_t ldc,
  int64_t batch_stride_C,
  void const * ptr_D,
  int64_t ldd,
  int64_t batch_stride_D,
  int max_alignment_in_bytes = 16
) {

  void const *pointers[] = {
    ptr_A, ptr_B, ptr_C, ptr_D
  };

  int64_t extents[] = {
    M, N, K, lda, ldb, ldc, ldd, batch_stride_A, batch_stride_B, batch_stride_C, batch_stride_D
  };

  NumericTypeID elements[] = {
    element_A, element_B, element_C
  };

  for (; max_alignment_in_bytes > 0; max_alignment_in_bytes /= 2) {
    
    bool satisfied = true;

    // Can pointers satisfy this?
    for (void const *ptr : pointers) {
      std::uintptr_t int_ptr = reinterpret_cast<std::uintptr_t>(ptr);

      if (int_ptr % max_alignment_in_bytes) {
        satisfied = false;
        break;
      }
    }

    if (!satisfied) {
      continue;
    }

    // Compute the maximum alignment based on element data types
    int max_element_alignment = 0;

    for (NumericTypeID type_id : elements) {
      int element_alignment = max_alignment_in_bytes * 8 / library::sizeof_bits(type_id); 
      max_element_alignment = std::max(max_element_alignment, element_alignment);
    }

    // Can the problem size and leading dimensions satisfy this?
    for (int64_t extent : extents) {
      if (extent % max_element_alignment) {
        satisfied = false;
        break;
      }
    }

    if (!satisfied) {
      continue;
    }

    // Yes
    return max_element_alignment;
  }

  // No alignment satisfies this problem
  return 0;
}

/// Find the best kernel in descending order of preference.
static Operation const * find_gemm_operation(
  GemmOperationFunctionalMap::const_iterator operators_it, 
  GemmPreferenceKey const preference_key) {

  auto cc_it = operators_it->second.upper_bound(preference_key);

  if (cc_it == operators_it->second.begin()) {
    return nullptr;
  }

  Operation const *operation = nullptr;

  // Search in descending order of compute capability
  do {
    --cc_it;

    // Search tile sizes in order, for now.
    for (auto const * op : cc_it->second) {

      GemmDescription const &desc = static_cast<GemmDescription const &>(op->description());

      int min_cc = desc.tile_description.minimum_compute_capability;
      int max_cc = desc.tile_description.maximum_compute_capability;

      int op_alignment = maximum_alignment_requirement(desc);

      if ((min_cc <= preference_key.compute_capability) &&
        (preference_key.compute_capability <= max_cc) &&
        (op_alignment <= preference_key.alignment)) {

        operation = op;
        break;
      }
    }
  } while (!operation && cc_it != operators_it->second.begin());

  return operation;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Executes a GEMM computation: D <= alpha * A*B + beta * C
Status Handle::gemm(

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix - ignored for real-valued matrices

  void const * ptr_A,                       /// Pointer to A matrix in Global Memory
  int64_t lda,                              /// Leading dimension of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix - ignored for real-valued matrices

  void const * ptr_B,                       /// Pointer to B matrix in Global Memory
  int64_t ldb,                              /// Leading dimension of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrices

  void const * ptr_C,                       /// Pointer to C matrix
  int64_t ldc,                              /// Leading dimension of C matrix

  void * ptr_D,                             /// Pointer to D matrix
  int64_t ldd                               /// Leading dimension of D matrix
) {
  
  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kGemm,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = gemm_problem_alignment(
    M, N, K, 
    element_A, ptr_A, lda, 0,
    element_B, ptr_B, ldb, 0,
    element_C, ptr_C, ldc, 0,
    ptr_D, ldd, 0, kMaximumAlignmentSize
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmConfiguration configuration{
    {M, N, K},
    lda,
    ldb,
    ldc,
    ldd,
    1
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmArguments arguments{
    ptr_A,
    ptr_B,
    ptr_C,
    ptr_D,
    alpha,
    beta,
    scalar_pointer_mode_
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Executes a GEMM computation: D <= alpha * A*B + beta * C.
//
// Supports batched-strided, batched array or split-K serial or split-K parallel.
//
Status Handle::gemm_universal(

  GemmUniversalMode mode,                   /// indicates the mode in which the kUniversal GEMM is launched

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix - ignored for real-valued matrices

  void const * ptr_A,                       /// Pointer to A matrix in Global Memory
  int64_t lda,                                  /// Leading dimension of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix - ignored for real-valued matrices

  void const * ptr_B,                       /// Pointer to B matrix in Global Memory
  int64_t ldb,                                  /// Leading dimension of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrices

  void const * ptr_C,                       /// Pointer to C matrix
  int64_t ldc,                                  /// Leading dimension of C matrix

  void * ptr_D,                             /// Pointer to D matrix
  int64_t ldd,                                  /// Leading dimension of D matrix

  int batch_count,                          /// Batch count or number of split-K slices

  int64_t batch_stride_A,                   /// Batch stride of A operand
  int64_t batch_stride_B,                   /// Batch stride of B operand
  int64_t batch_stride_C,                   /// Batch stride of C operand
  int64_t batch_stride_D                    /// Batch stride of D operand
) {
  
  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kUniversal,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  void const *ptr_A_check = ptr_A;
  void const *ptr_B_check = ptr_B;
  void const *ptr_C_check = ptr_C;
  void *      ptr_D_check = ptr_D;

  // Ignore alignment of pointers to pointers. We can't check this from the host,
  // as each batch index has its own pointer in device memory.
  if (mode == GemmUniversalMode::kArray) {
    ptr_A_check = nullptr; 
    ptr_B_check = nullptr; 
    ptr_C_check = nullptr; 
    ptr_D_check = nullptr; 
  }

  int alignment = gemm_problem_alignment(
    M, N, K, 
    element_A, ptr_A_check, lda, 0,
    element_B, ptr_B_check, ldb, 0,
    element_C, ptr_C_check, ldc, 0,
    ptr_D_check, ldd, 0, kMaximumAlignmentSize
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmUniversalConfiguration configuration{
    mode,
    {M, N, K},
    batch_count,
    lda,
    ldb,
    ldc,
    ldd
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  GemmUniversalArguments arguments{
    ptr_A,
    ptr_B,
    ptr_C,
    ptr_D,
    alpha,
    beta,
    scalar_pointer_mode_,
    batch_stride_A,
    batch_stride_B,
    batch_stride_C,
    batch_stride_D
  };

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration, &arguments);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Planar complex GEMM
Status Handle::gemm_planar_complex(

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix

  void const * ptr_A_real,                  /// Pointer to real part of A matrix
  void const * ptr_A_imag,                  /// Pointer to imaginary part of A matrix
  int64_t lda_real,                         /// Leading dimension of real part of A matrix
  int64_t lda_imag,                         /// Leading dimension of imaginary part of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix

  void const * ptr_B_real,                  /// Pointer to real part of B matrix
  void const * ptr_B_imag,                  /// Pointer to imaginary part of B matrix
  int64_t ldb_real,                             /// Leading dimension of real part of B matrix
  int64_t ldb_imag,                             /// Leading dimension of imaginary part of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrix

  void const * ptr_C_real,                  /// Pointer to real part of C matrix
  void const * ptr_C_imag,                  /// Pointer to imaginary part of C matrix
  int64_t ldc_real,                             /// Leading dimension of real part of C matrix
  int64_t ldc_imag,                             /// Leading dimension of imaginary part of C matrix

  void * ptr_D_real,                        /// Pointer to real part of D matrix
  void * ptr_D_imag,                        /// Pointer to imaginary part of D matrix
  int64_t ldd_real,                             /// Leading dimension of real part of D matrix
  int64_t ldd_imag,                             /// Leading dimension of imaginary part of D matrix

  int batch_count,                          /// Number of batched GEMMs to execute

  int64_t batch_stride_A_real,
  int64_t batch_stride_A_imag,

  int64_t batch_stride_B_real,
  int64_t batch_stride_B_imag,

  int64_t batch_stride_C_real,
  int64_t batch_stride_C_imag,

  int64_t batch_stride_D_real,
  int64_t batch_stride_D_imag
) {

  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kPlanarComplex,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = std::max(
    gemm_problem_alignment(
      M, N, K, 
      element_A, ptr_A_real, lda_real, batch_stride_A_real,
      element_B, ptr_B_real, ldb_real, batch_stride_B_real,
      element_C, ptr_C_real, ldc_real, batch_stride_C_real,
      ptr_D_real, ldd_real, batch_stride_D_real, kMaximumAlignmentSize
    ),
    gemm_problem_alignment(
      M, N, K, 
      element_A, ptr_A_imag, lda_imag, batch_stride_A_imag,
      element_B, ptr_B_imag, ldb_imag, batch_stride_B_imag,
      element_C, ptr_C_imag, ldc_imag, batch_stride_C_imag,
      ptr_D_imag, ldd_imag, batch_stride_D_imag, kMaximumAlignmentSize
    )
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmPlanarComplexConfiguration configuration{
    GemmUniversalMode::kBatched,
    {M, N, K},
    batch_count,
    lda_real,
    lda_imag,
    ldb_real,
    ldb_imag,
    ldc_real,
    ldc_imag,
    ldd_real,
    ldd_imag
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmPlanarComplexArguments arguments{
    ptr_A_real,
    ptr_A_imag,
    ptr_B_real,
    ptr_B_imag,
    ptr_C_real,
    ptr_C_imag,
    ptr_D_real,
    ptr_D_imag,
    alpha,
    beta,
    scalar_pointer_mode_,
    batch_stride_A_real,
    batch_stride_A_imag,
    batch_stride_B_real,
    batch_stride_B_imag,
    batch_stride_C_real,
    batch_stride_C_imag,
    batch_stride_D_real,
    batch_stride_D_imag
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Planar complex batched GEMM loading pointers from arrays in global memory
Status Handle::gemm_planar_complex_array(

  int expected_M,                           /// Expected GEMM M dimension (used for sizing CUDA grid)
  int expected_N,                           /// Expected GEMM N dimension (used for sizing CUDA grid)
  int expected_K,                           /// Expected GEMM K dimension
  int batch_count,                          /// Number of independent GEMM computations to execute

  int const *M,                             /// Array containing the GEMM M dimension for each batch index
  int const *N,                             /// Array containing the GEMM N dimension for each batch index
  int const *K,                             /// Array containing the GEMM K dimension for each batch index

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix

  void const * const * ptr_A_real,          /// Pointer to array containing pointers to real part of A matrices
  void const * const * ptr_A_imag,          /// Pointer to array containing pointers to imaginary part of A matrices

  int64_t lda_real,                             /// Leading dimension of real part of A matrix
  int64_t lda_imag,                             /// Leading dimension of imaginary part of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix

  void const * const * ptr_B_real,          /// Pointer to array containing pointers to real part of B matrices
  void const * const * ptr_B_imag,          /// Pointer to array containing pointers to imaginary part of B matrices

  int64_t ldb_real,                             /// Leading dimension of real part of B matrix
  int64_t ldb_imag,                             /// Leading dimension of imaginary part of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrix

  void const * const * ptr_C_real,          /// Pointer to array containing pointers to real part of C matrices
  void const * const * ptr_C_imag,          /// Pointer to array containing poitners to imaginary part of C matrices

  int64_t ldc_real,                             /// Leading dimension of real part of C matrix
  int64_t ldc_imag,                             /// Leading dimension of imaginary part of C matrix

  void * const * ptr_D_real,                /// Pointer to array containing pointers to real part of D matrices
  void * const * ptr_D_imag,                /// Pointer to array containing poitners to imaginary part of D matrices

  int64_t ldd_real,                             /// Leading dimension of real part of D matrix
  int64_t ldd_imag                              /// Leading dimension of imaginary part of D matrix
) {
  
  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kPlanarComplexArray,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = std::max(
    gemm_problem_alignment(
      expected_M, expected_N, expected_K, 
      element_A, nullptr, lda_real, 0,
      element_B, nullptr, ldb_real, 0,
      element_C, nullptr, ldc_real, 0,
      nullptr, ldd_real, 0, kMaximumAlignmentSize
    ),
    gemm_problem_alignment(
      expected_M, expected_N, expected_K, 
      element_A, nullptr, lda_imag, 0,
      element_B, nullptr, ldb_imag, 0,
      element_C, nullptr, ldc_imag, 0,
      nullptr, ldd_imag, 0, kMaximumAlignmentSize
    )
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmPlanarComplexArrayConfiguration configuration{
    {expected_M, expected_N, expected_K},
    batch_count,
    lda_real,
    lda_imag,
    ldb_real,
    ldb_imag,
    ldc_real,
    ldc_imag,
    ldd_real,
    ldd_imag
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmPlanarComplexArrayArguments arguments{
    M, N, K,
    ptr_A_real,
    ptr_A_imag,
    ptr_B_real,
    ptr_B_imag,
    ptr_C_real,
    ptr_C_imag,
    ptr_D_real,
    ptr_D_imag,
    alpha,
    beta,
    scalar_pointer_mode_
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Finds conv operation instances with Conv::ElementC = Reduction::ElementWorkspace
Operation const* find_conv_operation_for_parallel_reduction(Operation const *operation) {

  ConvDescription const &conv_desc = 
    static_cast<ConvDescription const &>(operation->description());

  // if the curren conv operation accumulator and output data type match return operation
  if(conv_desc.tile_description.math_instruction.element_accumulator == conv_desc.C.element) {
    return operation;
  }

  // find conv operation to match conv output and reduction workspace data type
  ConvFunctionalKey key(
    library::Provider::kCUTLASS,
    conv_desc.conv_kind,        
    conv_desc.A.element,
    conv_desc.A.layout,
    conv_desc.B.element,
    conv_desc.B.layout,
    conv_desc.tile_description.math_instruction.element_accumulator,
    conv_desc.C.layout,
    conv_desc.tile_description.math_instruction.element_accumulator, 
    conv_desc.element_epilogue);

  // conv operation table for conv2d or conv3d
  auto conv_operations = (conv_desc.kind == OperationKind::kConv2d) ? 
                          Singleton::get().operation_table.conv2d_operations : 
                          Singleton::get().operation_table.conv3d_operations;

  // find ConvFunctionalKey in convolution operation table
  auto operators_it = conv_operations.find(key);

  if (operators_it == conv_operations.end()) {
    return nullptr;
  }
  
  if (operators_it->second.empty()) {
    return nullptr;
  }

  // conv operation for same compute capability and iterator algorithm
  ConvPreferenceKey preference_key(
    conv_desc.tile_description.minimum_compute_capability, 
    conv_desc.iterator_algorithm);

  auto it = operators_it->second.find(preference_key);
  
  if(it == operators_it->second.end()) {
    return nullptr;
  }

  // return matching conv opertion (same tile sizes and instruction)
  for (auto op : it->second) {
    if (op->description().tile_description == operation->description().tile_description) {
      return op;
    }
  }

  return nullptr;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Finds gemm operation instances with Gemm::ElementC = Reduction::ElementWorkspace
Operation const* find_gemm_operation_for_parallel_reduction(Operation const *operation) {

  GemmDescription const &gemm_desc = 
    static_cast<GemmDescription const &>(operation->description());

  // if the curren gemm operation accumulator and output data type match return operation
  if(gemm_desc.tile_description.math_instruction.element_accumulator == gemm_desc.C.element) {
    return operation;
  }

  // find gemm operation to match gemm output and reduction workspace data type
  GemmFunctionalKey key(
    library::Provider::kCUTLASS,
    gemm_desc.gemm_kind,
    gemm_desc.tile_description.math_instruction.element_accumulator,
    gemm_desc.element_epilogue,
    gemm_desc.A.element,
    gemm_desc.A.layout,
    gemm_desc.transform_A,
    gemm_desc.B.element,
    gemm_desc.B.layout,
    gemm_desc.transform_B,
    gemm_desc.tile_description.math_instruction.element_accumulator);

  // gemm operation table
  auto gemm_operations = Singleton::get().operation_table.gemm_operations;

  // find ConvFunctionalKey in gemm operation table
  auto operators_it = gemm_operations.find(key);

  if (operators_it == gemm_operations.end()) {
    return nullptr;
  }

  if (operators_it->second.empty()) {
    return nullptr;
  }

  // A and B uses the same alignment in the generator.py
  int alignment = gemm_desc.A.alignment;

  // gemm operation for same compute capability and iterator algorithm
  GemmPreferenceKey preference_key(
    gemm_desc.tile_description.minimum_compute_capability, 
    alignment);

  return find_gemm_operation(operators_it, preference_key);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace library
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
