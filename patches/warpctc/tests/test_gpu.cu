#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cmath>
#include <iostream>
#include <random>
#include <tuple>
#include <vector>

#include <ctc.h>

#include "test.h"

bool small_test() {
  const int alphabet_size = 5;
  const int T = 2;

  std::vector<float> activations = {
      0.1f, 0.6f, 0.1f, 0.1f, 0.1f, 0.1f, 0.1f, 0.6f, 0.1f, 0.1f};

  // Calculate the score analytically
  float expected_score;
  {
    std::vector<float> probs(activations.size());
    softmax(activations.data(), alphabet_size, T, probs.data());

    // Score calculation is specific to the given activations above
    expected_score = probs[1] * probs[7];
  }

  hipStream_t stream;
  throw_on_error(hipStreamCreate(&stream), "hipStreamCreate");

  float *activations_gpu;
  throw_on_error(
      hipMalloc(&activations_gpu, activations.size() * sizeof(float)),
      "hipMalloc");
  throw_on_error(hipMemcpyAsync(activations_gpu,
                                 activations.data(),
                                 activations.size() * sizeof(float),
                                 hipMemcpyHostToDevice,
                                 stream),
                 "hipMemcpyAsync");

  std::vector<int> labels = {1, 2};
  std::vector<int> label_lengths = {2};

  std::vector<int> lengths;
  lengths.push_back(T);

  float score;

  ctcOptions options{};
  options.loc = CTC_GPU;
  options.stream = stream;

  size_t gpu_alloc_bytes;
  throw_on_error(get_workspace_size(label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    options,
                                    &gpu_alloc_bytes),
                 "Error: get_workspace_size in small_test");

  char *ctc_gpu_workspace;
  throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes), "hipMalloc");

  throw_on_error(compute_ctc_loss(activations_gpu,
                                  nullptr,
                                  labels.data(),
                                  label_lengths.data(),
                                  lengths.data(),
                                  alphabet_size,
                                  lengths.size(),
                                  &score,
                                  ctc_gpu_workspace,
                                  options),
                 "Error: compute_ctc_loss in small_test");

  score = std::exp(-score);
  const float eps = 1e-6;

  const float lb = expected_score - eps;
  const float ub = expected_score + eps;

  throw_on_error(hipFree(activations_gpu), "hipFree");
  throw_on_error(hipFree(ctc_gpu_workspace), "hipFree");
  throw_on_error(hipStreamDestroy(stream), "hipStreamDestroy");

  return (score > lb && score < ub);
}

int offset(int t, int n, int a) {
  constexpr int minibatch = 2;
  constexpr int alphabet_size = 6;
  return (t * minibatch + n) * alphabet_size + a;
}

bool options_test() {
  const int alphabet_size = 6;
  const int T = 5;
  const int minibatch = 2;

  std::vector<float> activations = {
      0.633766f,  0.221185f, 0.0917319f, 0.0129757f,  0.0142857f,  0.0260553f,
      0.30176f,   0.28562f,  0.0831517f, 0.0862751f,  0.0816851f,  0.161508f,

      0.111121f,  0.588392f, 0.278779f,  0.0055756f,  0.00569609f, 0.010436f,
      0.24082f,   0.397533f, 0.0557226f, 0.0546814f,  0.0557528f,  0.19549f,

      0.0357786f, 0.633813f, 0.321418f,  0.00249248f, 0.00272882f, 0.0037688f,
      0.230246f,  0.450868f, 0.0389607f, 0.038309f,   0.0391602f,  0.202456f,

      0.0663296f, 0.643849f, 0.280111f,  0.00283995f, 0.0035545f,  0.00331533f,
      0.280884f,  0.429522f, 0.0326593f, 0.0339046f,  0.0326856f,  0.190345f,

      0.458235f,  0.396634f, 0.123377f,  0.00648837f, 0.00903441f, 0.00623107f,
      0.423286f,  0.315517f, 0.0338439f, 0.0393744f,  0.0339315f,  0.154046f};

  std::vector<float> expected_grads =  // from tensorflow
      {-0.366234f,  0.221185f,   0.0917319f, 0.0129757f,
       0.0142857f,  0.0260553f,  -0.69824f,  0.28562f,
       0.0831517f,  0.0862751f,  0.0816851f, 0.161508f,

       0.111121f,   -0.411608f,  0.278779f,  0.0055756f,
       0.00569609f, 0.010436f,   0.24082f,   -0.602467f,
       0.0557226f,  0.0546814f,  0.0557528f, 0.19549f,

       0.0357786f,  0.633813f,   -0.678582f, 0.00249248f,
       0.00272882f, 0.0037688f,  0.230246f,  0.450868f,
       0.0389607f,  0.038309f,   0.0391602f, -0.797544f,

       0.0663296f,  -0.356151f,  0.280111f,  0.00283995f,
       0.0035545f,  0.00331533f, 0.280884f,  -0.570478f,
       0.0326593f,  0.0339046f,  0.0326856f, 0.190345f,

       -0.541765f,  0.396634f,   0.123377f,  0.00648837f,
       0.00903441f, 0.00623107f, -0.576714f, 0.315517f,
       0.0338439f,  0.0393744f,  0.0339315f, 0.154046f};

  // Calculate the expected scores analytically
  auto &a = activations;
  double expected_score[2];
  expected_score[0] =
      -std::log(a[offset(0, 0, 0)] * a[offset(1, 0, 1)] * a[offset(2, 0, 2)] *
                a[offset(3, 0, 1)] * a[offset(4, 0, 0)]);
  expected_score[1] = 5.42262f;  // from tensorflow

  // now take the log to account for the softmax
  for (auto &a : activations) {
    a = std::log(a);
  }

  hipStream_t stream;
  throw_on_error(hipStreamCreate(&stream), "hipStreamCreate");

  float *activations_gpu;
  throw_on_error(
      hipMalloc(&activations_gpu, activations.size() * sizeof(float)),
      "hipMalloc");
  throw_on_error(hipMemcpyAsync(activations_gpu,
                                 activations.data(),
                                 activations.size() * sizeof(float),
                                 hipMemcpyHostToDevice,
                                 stream),
                 "hipMemcpyAsync");

  std::vector<int> labels = {0, 1, 2, 1, 0, 0, 1, 1, 0};

  std::vector<int> label_lengths = {5, 4};

  std::vector<int> lengths = {5, 5};

  float score[2];

  float *grads_gpu;
  throw_on_error(
      hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
      "hipMalloc");

  ctcOptions options{};
  options.loc = CTC_GPU;
  options.stream = stream;
  options.blank_label = 5;

  size_t gpu_alloc_bytes;
  throw_on_error(get_workspace_size(label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    options,
                                    &gpu_alloc_bytes),
                 "Error: get_workspace_size in options_test");

  char *ctc_gpu_workspace;
  throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes), "hipMalloc");

  throw_on_error(compute_ctc_loss(activations_gpu,
                                  grads_gpu,
                                  labels.data(),
                                  label_lengths.data(),
                                  lengths.data(),
                                  alphabet_size,
                                  lengths.size(),
                                  &score[0],
                                  ctc_gpu_workspace,
                                  options),
                 "Error: compute_ctc_loss in options_test");

  std::vector<float> grads(alphabet_size * T * minibatch);
  throw_on_error(hipMemcpyAsync(grads.data(),
                                 grads_gpu,
                                 grads.size() * sizeof(float),
                                 hipMemcpyDeviceToHost,
                                 stream),
                 "hipMemcpyAsync");
  throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

  throw_on_error(hipFree(activations_gpu), "hipFree");
  throw_on_error(hipFree(ctc_gpu_workspace), "hipFree");
  throw_on_error(hipStreamDestroy(stream), "hipStreamDestroy");

  const double eps = 1e-4;

  bool result = true;
  for (int i = 0; i < grads.size(); i++) {
    const double lb = expected_grads[i] - eps;
    const double ub = expected_grads[i] + eps;
    if (!(grads[i] > lb && grads[i] < ub)) {
      std::cerr << "grad mismatch in options_test"
                << " expected grad: " << expected_grads[i]
                << " calculated score: " << grads[i] << " !(" << lb << " < "
                << grads[i] << " < " << ub << ")" << std::endl;
      result = false;
    }
  }

  for (int i = 0; i < 2; i++) {
    const double lb = expected_score[i] - eps;
    const double ub = expected_score[i] + eps;

    if (!(score[i] > lb && score[i] < ub)) {
      std::cerr << "score mismatch in options_test"
                << " expected score: " << expected_score[i]
                << " calculated score: " << score[i] << std::endl;
      result = false;
    }
  }
  return result;
}

bool inf_test() {
  const int alphabet_size = 15;
  const int T = 50;
  const int L = 10;
  const int minibatch = 1;

  std::vector<int> labels = genLabels(alphabet_size, L);
  labels[0] = 2;
  std::vector<int> label_lengths = {L};

  std::vector<float> acts = genActs(alphabet_size * T * minibatch);

  for (int i = 0; i < T; ++i) acts[alphabet_size * i + 2] = -1e30;

  hipStream_t stream;
  throw_on_error(hipStreamCreate(&stream), "hipStreamCreate");

  float *acts_gpu;
  throw_on_error(hipMalloc(&acts_gpu, acts.size() * sizeof(float)),
                 "hipMalloc");
  throw_on_error(hipMemcpyAsync(acts_gpu,
                                 acts.data(),
                                 acts.size() * sizeof(float),
                                 hipMemcpyHostToDevice,
                                 stream),
                 "hipMemcpyAsync");

  std::vector<int> lengths;
  lengths.push_back(T);

  float *grads_gpu;
  throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T) * sizeof(float)),
                 "hipMalloc");

  float cost;

  ctcOptions options{};
  options.loc = CTC_GPU;
  options.stream = stream;

  size_t gpu_alloc_bytes;
  throw_on_error(get_workspace_size(label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    options,
                                    &gpu_alloc_bytes),
                 "Error: get_workspace_size in inf_test");

  char *ctc_gpu_workspace;
  throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes), "hipMalloc");

  throw_on_error(compute_ctc_loss(acts_gpu,
                                  grads_gpu,
                                  labels.data(),
                                  label_lengths.data(),
                                  lengths.data(),
                                  alphabet_size,
                                  lengths.size(),
                                  &cost,
                                  ctc_gpu_workspace,
                                  options),
                 "Error: compute_ctc_loss in inf_test");

  bool status = std::isinf(cost);

  std::vector<float> grads(alphabet_size * T);
  throw_on_error(hipMemcpyAsync(grads.data(),
                                 grads_gpu,
                                 grads.size() * sizeof(float),
                                 hipMemcpyDeviceToHost,
                                 stream),
                 "hipMemcpyAsync");
  throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

  for (int i = 0; i < alphabet_size * T; ++i) status &= !std::isnan(grads[i]);

  throw_on_error(hipFree(acts_gpu), "hipFree");
  throw_on_error(hipFree(grads_gpu), "hipFree");
  throw_on_error(hipFree(ctc_gpu_workspace), "hipFree");
  throw_on_error(hipStreamDestroy(stream), "hipStreamDestroy");

  return status;
}

float grad_check(int T,
                 int alphabet_size,
                 std::vector<float> &acts,
                 const std::vector<std::vector<int>> &labels,
                 const std::vector<int> &lengths) {
  float epsilon = 1e-2;

  const int minibatch = labels.size();

  hipStream_t stream;
  throw_on_error(hipStreamCreate(&stream), "hipStreamCreate");

  float *acts_gpu;
  throw_on_error(hipMalloc(&acts_gpu, acts.size() * sizeof(float)),
                 "hipMalloc");
  throw_on_error(hipMemcpyAsync(acts_gpu,
                                 acts.data(),
                                 acts.size() * sizeof(float),
                                 hipMemcpyHostToDevice,
                                 stream),
                 "hipMemcpyAsync");

  std::vector<int> flat_labels;
  std::vector<int> label_lengths;
  for (const auto &l : labels) {
    flat_labels.insert(flat_labels.end(), l.begin(), l.end());
    label_lengths.push_back(l.size());
  }

  std::vector<float> costs(minibatch);

  float *grads_gpu;
  throw_on_error(hipMalloc(&grads_gpu, acts.size() * sizeof(float)),
                 "hipMalloc");

  ctcOptions options{};
  options.loc = CTC_GPU;
  options.stream = stream;

  size_t gpu_alloc_bytes;
  throw_on_error(get_workspace_size(label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    options,
                                    &gpu_alloc_bytes),
                 "Error: get_workspace_size in grad_check");

  char *ctc_gpu_workspace;
  throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes), "hipMalloc");

  throw_on_error(compute_ctc_loss(acts_gpu,
                                  grads_gpu,
                                  flat_labels.data(),
                                  label_lengths.data(),
                                  lengths.data(),
                                  alphabet_size,
                                  minibatch,
                                  costs.data(),
                                  ctc_gpu_workspace,
                                  options),
                 "Error: compute_ctc_loss (0) in grad_check");

  std::vector<float> grads(acts.size());
  throw_on_error(hipMemcpyAsync(grads.data(),
                                 grads_gpu,
                                 grads.size() * sizeof(float),
                                 hipMemcpyDeviceToHost,
                                 stream),
                 "hipMemcpyAsync");
  throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");
  std::vector<float> num_grad(grads.size());

  // perform 2nd order central differencing
  for (int i = 0; i < T * alphabet_size * minibatch; ++i) {
    acts[i] += epsilon;

    throw_on_error(hipMemcpyAsync(acts_gpu,
                                   acts.data(),
                                   acts.size() * sizeof(float),
                                   hipMemcpyHostToDevice,
                                   stream),
                   "hipMemcpyAsync");

    std::vector<float> costsP1(minibatch);
    std::vector<float> costsP2(minibatch);

    throw_on_error(compute_ctc_loss(acts_gpu,
                                    NULL,
                                    flat_labels.data(),
                                    label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    minibatch,
                                    costsP1.data(),
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss (1) in grad_check");

    acts[i] -= 2 * epsilon;
    throw_on_error(hipMemcpyAsync(acts_gpu,
                                   acts.data(),
                                   acts.size() * sizeof(float),
                                   hipMemcpyHostToDevice,
                                   stream),
                   "hipMemcpyAsync");

    throw_on_error(compute_ctc_loss(acts_gpu,
                                    NULL,
                                    flat_labels.data(),
                                    label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    minibatch,
                                    costsP2.data(),
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss (2) in grad_check");

    float costP1 = std::accumulate(costsP1.begin(), costsP1.end(), 0.);
    float costP2 = std::accumulate(costsP2.begin(), costsP2.end(), 0.);

    acts[i] += epsilon;

    num_grad[i] = (costP1 - costP2) / (2 * epsilon);
  }

  float diff = rel_diff(grads, num_grad);

  throw_on_error(hipFree(acts_gpu), "hipFree");
  throw_on_error(hipFree(grads_gpu), "hipFree");
  throw_on_error(hipFree(ctc_gpu_workspace), "hipFree");
  throw_on_error(hipStreamDestroy(stream), "hipStreamDestroy");

  return diff;
}

bool run_tests() {
  std::vector<std::tuple<int, int, int, int, float>> problem_sizes = {
      std::make_tuple(28, 50, 15, 1, 1e-5)};

  bool status = true;
  for (auto problem : problem_sizes) {
    int alphabet_size, T, L, minibatch;
    float tol;
    std::tie(alphabet_size, T, L, minibatch, tol) = problem;

    std::vector<float> acts = genActs(alphabet_size * T * minibatch);

    std::vector<std::vector<int>> labels;
    std::vector<int> sizes;
    for (int mb = 0; mb < minibatch; ++mb) {
      int actual_length = L;
      labels.push_back(genLabels(alphabet_size, actual_length));
      sizes.push_back(T);
    }

    float diff = grad_check(T, alphabet_size, acts, labels, sizes);
    status &= (diff < tol);
  }

  return status;
}

int main(void) {
  if (get_warpctc_version() != 2) {
    std::cerr << "Invalid WarpCTC version." << std::endl;
    return 1;
  }

  std::cout << "Running GPU tests" << std::endl;
  throw_on_error(hipSetDevice(0), "hipSetDevice");

  bool status = true;
  status &= small_test();
  status &= options_test();
  status &= inf_test();
  status &= run_tests();

  if (status) {
    std::cout << "Tests pass" << std::endl;
    return 0;
  } else {
    std::cout << "Some or all tests fail" << std::endl;
    return 1;
  }
}
