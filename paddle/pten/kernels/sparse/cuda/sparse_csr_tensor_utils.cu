/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/sparse.h"
#include "paddle/pten/api/lib/utils/allocator.h"
#include "paddle/pten/core/tensor_meta.h"
#include "paddle/pten/kernels/gpu/utils.h"
#include "paddle/pten/kernels/sparse/cuda/sparse_csr_tensor_utils.h"

namespace pten {

template <typename T>
void ToSparseCsr(const HIPContext& dev_ctx,
                 const DenseTensor& src,
                 SparseCsrTensor* dst) {
  PADDLE_ENFORCE_EQ(src.dims().size(),
                    2,
                    paddle::platform::errors::InvalidArgument(
                        "SparseCsrTensor only support 2-D Tensor."));

  const T* src_data = src.data<T>();
  const auto& src_dims = src.dims();

  const auto cpu_alloc =
      std::make_shared<paddle::experimental::DefaultAllocator>(
          paddle::platform::CPUPlace());
  const auto allocator =
      std::make_shared<paddle::experimental::DefaultAllocator>(src.place());
  auto nnz_dims = paddle::framework::make_ddim({src_dims[0] + 1});
  DenseTensorMeta nnz_meta(DataType::INT32, nnz_dims, DataLayout::NCHW);
  DenseTensor nnz_tensor(allocator, nnz_meta);
  DenseTensor cpu_nnz_tensor(cpu_alloc, nnz_meta);

  auto sparse =
      paddle::operators::math::GetSparse<paddle::platform::CUDADeviceContext,
                                         T>(dev_ctx);
  int* nnz = nnz_tensor.mutable_data<int32_t>();
  const int M = static_cast<int>(src_dims[0]);
  const int N = static_cast<int>(src_dims[1]);
  sparse.nnz(M, N, src_data, nnz, nnz + 1);
  pten::Copy(dev_ctx, nnz_tensor, true, &cpu_nnz_tensor);
  const int64_t non_zero_num = cpu_nnz_tensor.data<int>()[0];

  dst->Resize(src_dims, non_zero_num);

  int64_t* crows_data = dst->mutable_non_zero_crows();
  int64_t* cols_data = dst->mutable_non_zero_cols();
  T* values_data = dst->mutable_non_zero_elements<T>();
  sparse.DenseToSparseCsr(static_cast<int>(src_dims[0]),
                          static_cast<int>(src_dims[1]),
                          src_data,
                          crows_data,
                          cols_data,
                          values_data);
}

template <typename T>
void SparseCsrToDense(const HIPContext& dev_ctx,
                      const SparseCsrTensor& src,
                      DenseTensor* dst) {
  auto sparse =
      paddle::operators::math::GetSparse<paddle::platform::CUDADeviceContext,
                                         T>(dev_ctx);
  const auto src_dims = src.dims();
  const int M = src_dims[0];
  const int N = src_dims[1];
  const DenseTensor& crows = src.non_zero_crows();
  const DenseTensor& cols = src.non_zero_cols();
  const DenseTensor& values = src.non_zero_elements();
  const int64_t nnz = src.nnz();
  sparse.SparseCsrToDense(M,
                          N,
                          nnz,
                          crows.data<int64_t>(),
                          cols.data<int64_t>(),
                          values.data<T>(),
                          dst->mutable_data<T>());
}

}  // namespace pten

PT_REGISTER_KERNEL(
    to_sparse_csr, GPU, ALL_LAYOUT, pten::ToSparseCsr, float, double) {}
PT_REGISTER_KERNEL(sparse_csr_to_dense,
                   GPU,
                   ALL_LAYOUT,
                   pten::SparseCsrToDense,
                   float,
                   double) {}
