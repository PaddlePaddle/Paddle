/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/sparse.h"
#include "paddle/pten/api/lib/utils/allocator.h"
#include "paddle/pten/core/tensor_meta.h"
#include "paddle/pten/kernels/gpu/utils.h"
#include "paddle/pten/kernels/hybird/sparse/cuda/sparse_utils.h"
#include "paddle/pten/kernels/sparse/cuda/sparse_csr_tensor_utils.h"

namespace pten {

template <typename T>
void ToSparseCsr(const HIPContext& dev_ctx,
                 const DenseTensor& src,
                 SparseCsrTensor* dst) {
  PADDLE_ENFORCE_EQ(src.dims().size(),
                    2,
                    paddle::platform::errors::InvalidArgument(
                        "SparseCsrTensor only support 2-D Tensor."));

  const T* src_data = src.data<T>();
  const auto& src_dims = src.dims();

  const auto cpu_alloc =
      std::make_shared<paddle::experimental::DefaultAllocator>(
          paddle::platform::CPUPlace());
  const auto allocator =
      std::make_shared<paddle::experimental::DefaultAllocator>(src.place());
  auto nnz_dims = paddle::framework::make_ddim({src_dims[0] + 1});
  DenseTensorMeta nnz_meta(DataType::INT32, nnz_dims, DataLayout::NCHW);
  DenseTensor nnz_tensor(allocator, nnz_meta);
  DenseTensor cpu_nnz_tensor(cpu_alloc, nnz_meta);

  int* nnz = nnz_tensor.mutable_data<int32_t>();
  get_non_zero_num<T>(dev_ctx, src, 2, nnz, nnz + 1);
  pten::Copy(dev_ctx, nnz_tensor, true, &cpu_nnz_tensor);
  const int64_t non_zero_num = cpu_nnz_tensor.data<int>()[0];

  dst->Resize(src.meta(), non_zero_num);

  int64_t* crows_data = dst->mutable_non_zero_crows();
  int64_t* cols_data = dst->mutable_non_zero_cols();
  T* values_data = dst->mutable_non_zero_elements<T>();
  auto sparse =
      paddle::operators::math::GetSparse<paddle::platform::CUDADeviceContext,
                                         T>(dev_ctx);
  sparse.DenseToSparseCsr(static_cast<int>(src_dims[0]),
                          static_cast<int>(src_dims[1]),
                          src_data,
                          crows_data,
                          cols_data,
                          values_data);
}

}  // namespace pten

PT_REGISTER_KERNEL(
    to_sparse_csr, GPU, ALL_LAYOUT, pten::ToSparseCsr, float, double) {}
