#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/remove.h>

#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/core/tensor_meta.h"
#include "paddle/pten/kernels/sparse/sparse_utils_kernel.h"

namespace pten {
namespace sparse {

template <typename T>
inline __device__ bool DevIsZero(const T* data, const int64_t cols) {
  const T zero = static_cast<T>(0);
  // TODO(zhangkaihuo): check the data is zero or not in parallen when cols > 1
  for (int64_t i = 0; i < cols; i++) {
    if (data[i] != zero) {
      return false;
    }
  }
  return true;
}

template <typename T>
__global__ void GetNonZeroNums(const T* dense_data,
                               const int rows,
                               const int cols,
                               int* non_zero_num,
                               int* temp_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ int counter;
  if (threadIdx.x == 0) counter = 0;
  __syncthreads();

  for (int i = tid; i < rows; i += gridDim.x * blockDim.x) {
    int index = -1;
    // TODO(zhangkaihuo): when cols=1, vectorization can be used
    if (!DevIsZero(dense_data + i * cols, cols)) {
      // use reductions?
      atomicAdd(&counter, 1);
      index = i;
    }
    temp_indexs[i] = index;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(non_zero_num, counter);
  }
}

template <typename T>
__global__ void GetNonZeroElementsAndIndices(const T* dense_data,
                                             const int64_t sparse_dim,
                                             const int64_t cols,
                                             const int64_t* x_dims,
                                             const int non_zero_num,
                                             const int* indexs,
                                             int64_t* indices,
                                             T* sparse_data) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int64_t sparse_index = indexs[i];
    int64_t x_index = sparse_index;
    for (int64_t j = sparse_dim - 1; j >= 0; j--) {
      indices[j * non_zero_num + i] = sparse_index % x_dims[j];
      sparse_index /= x_dims[j];
    }

    for (int j = 0; j < cols; j++) {
      sparse_data[i * cols + j] = dense_data[x_index * cols + j];
    }
  }
}

template <typename Context>
void GetGpuLaunchConfig1D(const Context& dev_ctx,
                          const int64_t n,
                          int* grid_size,
                          int* block_size) {
  const int MAX_BLOCK_DIM = dev_ctx.GetMaxThreadsPerBlock();
  const int MAX_GRID_DIM = dev_ctx.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
  *block_size = (n >= MAX_BLOCK_DIM) ? MAX_BLOCK_DIM
                                     : (1 << static_cast<int>(std::log2(n)));
  *grid_size = n / *block_size;
  *grid_size = (*grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : *grid_size;
}

template <typename T, typename Context>
void DenseToSparseCooKernel(const Context& dev_ctx,
                            const DenseTensor& x,
                            const int64_t sparse_dim,
                            SparseCooTensor* out) {
  const T* x_data = x.data<T>();
  const auto& x_dims = x.dims();
  auto dims_2d = flatten_to_2d(x_dims, sparse_dim);
  const int rows = dims_2d[0];
  const int cols = dims_2d[1];
  auto nums_meta =
      pten::DenseTensorMeta(DataType::INT32, {1}, pten::DataLayout::NCHW);
  DenseTensor nums = pten::Empty(dev_ctx, std::move(nums_meta));
  auto x_dims_meta =
      pten::DenseTensorMeta(DataType::INT64,
                            {static_cast<int64_t>(x_dims.size())},
                            pten::DataLayout::NCHW);
  DenseTensor d_x_dims = pten::Empty(dev_ctx, std::move(x_dims_meta));

  const auto place = dev_ctx.GetPlace();

  // 1. get numbers of non zero elements, and get the index of non zero elements
  int* nums_ptr = nums.mutable_data<int>(place);
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(nums_ptr, 0, sizeof(int), dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(nums_ptr, 0, sizeof(int), dev_ctx.stream()));
#endif
  int grid_size = 1, block_size = 1;
  GetGpuLaunchConfig1D(dev_ctx, rows, &grid_size, &block_size);

  auto temp_indexs_meta =
      pten::DenseTensorMeta(DataType::INT32, {rows}, pten::DataLayout::NCHW);
  DenseTensor temp_indexs = pten::Empty(dev_ctx, std::move(temp_indexs_meta));
  int* temp_indexs_ptr = temp_indexs.mutable_data<int>(place);
  GetNonZeroNums<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
      x_data, rows, cols, nums_ptr, temp_indexs_ptr);
#ifdef PADDLE_WITH_HIP
  thrust::remove(thrust::hip::par.on(dev_ctx.stream()),
#else
  thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                 temp_indexs_ptr,
                 temp_indexs_ptr + rows,
                 -1);

  // 2. copy non_zero_num to host, copy x_dims to device
  int non_zero_num = 0;
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&non_zero_num,
                                            nums_ptr,
                                            sizeof(int),
                                            hipMemcpyDeviceToHost,
                                            dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&non_zero_num,
                                             nums_ptr,
                                             sizeof(int),
                                             hipMemcpyDeviceToHost,
                                             dev_ctx.stream()));
#endif

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemcpyAsync(d_x_dims.mutable_data<int64_t>(place),
                     x_dims.Get(),
                     x_dims.size() * sizeof(x_dims[0]),
                     hipMemcpyHostToDevice,
                     dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemcpyAsync(d_x_dims.mutable_data<int64_t>(place),
                      x_dims.Get(),
                      x_dims.size() * sizeof(x_dims[0]),
                      hipMemcpyHostToDevice,
                      dev_ctx.stream()));
#endif

  dev_ctx.Wait();  // wait the copy

  const auto values_dims = InferDenseDims(x_dims, sparse_dim, non_zero_num);
  DenseTensorMeta indices_meta(DataType::INT64,
                               {sparse_dim, static_cast<int64_t>(non_zero_num)},
                               DataLayout::NCHW);
  DenseTensorMeta values_meta(x.meta().dtype, values_dims, x.meta().layout);
  pten::DenseTensor indices(
      pten::make_intrusive<paddle::experimental::SharedStorage>(
          dev_ctx.GetPlace()),
      std::move(indices_meta));
  pten::DenseTensor values(
      pten::make_intrusive<paddle::experimental::SharedStorage>(
          dev_ctx.GetPlace()),
      std::move(values_meta));
  int64_t* indices_data = indices.mutable_data<int64_t>(place);
  T* sparse_data = values.mutable_data<T>(place);

  // 3. calc indices by indexs and get values by indexs
  GetGpuLaunchConfig1D(dev_ctx, non_zero_num, &grid_size, &block_size);
  GetNonZeroElementsAndIndices<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
      x_data,
      sparse_dim,
      cols,
      d_x_dims.data<int64_t>(),
      non_zero_num,
      temp_indexs_ptr,
      indices_data,
      sparse_data);
  out->SetMember(indices, values, x_dims, true);
}

__global__ void GetBatchSizes(const int64_t* crows,
                              const int rows,
                              const int batchs,
                              int* batch_sizes) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < batchs) {
    batch_sizes[tid] = crows[tid * (rows + 1) + rows];
  }
}

__global__ void ConvertCsrCrowsToCooRows(const int64_t* crows_ptr,
                                         const int* crows_offsets,
                                         int64_t* rows_ptr,
                                         int64_t* batch_ptr,
                                         const int rows) {
  const int b = blockIdx.y;
  const int64_t offset = crows_offsets ? crows_offsets[b] : 0;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < rows; i += gridDim.x * blockDim.x) {
    for (int j = crows_ptr[b * (rows + 1) + i];
         j < crows_ptr[b * (rows + 1) + i + 1];
         j++) {
      rows_ptr[offset + j] = i;
      if (batch_ptr) {
        batch_ptr[offset + j] = b;
      }
    }
  }
}

template <typename T, typename Context>
void SparseCsrToCooKernel(const Context& dev_ctx,
                          const SparseCsrTensor& x,
                          SparseCooTensor* out) {
  const DDim& x_dims = x.dims();
  const int64_t non_zero_num = x.non_zero_cols().numel();
  const auto& csr_crows = x.non_zero_crows();
  const auto& csr_cols = x.non_zero_cols();
  const auto& csr_values = x.non_zero_elements();
  const int64_t* csr_crows_data = csr_crows.data<int64_t>();
  const int64_t* csr_cols_data = csr_cols.data<int64_t>();
  const T* csr_values_data = csr_values.data<T>();

  int64_t sparse_dim = 2;
  if (x_dims.size() == 3) {
    sparse_dim = 3;
  }
  int batchs = x_dims.size() == 2 ? 1 : x_dims[0];
  int rows = x_dims.size() == 2 ? x_dims[0] : x_dims[1];

  const auto place = dev_ctx.GetPlace();
  DenseTensorMeta indices_meta(
      DataType::INT64, {sparse_dim, non_zero_num}, DataLayout::NCHW);
  DenseTensorMeta values_meta(x.dtype(), {non_zero_num}, x.layout());
  DenseTensorMeta offsets_meta(DataType::INT32, {batchs}, DataLayout::NCHW);
  DenseTensor indices = pten::Empty(dev_ctx, std::move(indices_meta));
  DenseTensor values = pten::Empty(dev_ctx, std::move(values_meta));
  DenseTensor offsets = pten::Empty(dev_ctx, std::move(offsets_meta));
  int64_t* coo_indices = indices.mutable_data<int64_t>(place);
  int64_t* batch_ptr = x_dims.size() == 2 ? nullptr : coo_indices;
  int64_t* coo_rows_data =
      x_dims.size() == 2 ? coo_indices : batch_ptr + non_zero_num;
  int64_t* coo_cols_data = coo_rows_data + non_zero_num;
  int* offsets_ptr = batchs == 1 ? nullptr : offsets.mutable_data<int>(place);
  T* coo_values_data = values.mutable_data<T>(place);

  int grid_size = 1, block_size = 1;
  if (batchs > 1) {
    GetGpuLaunchConfig1D(dev_ctx, batchs, &grid_size, &block_size);
    GetBatchSizes<<<grid_size, block_size>>>(
        csr_crows_data, rows, batchs, offsets_ptr);

#ifdef PADDLE_WITH_HIP
    thrust::exclusive_scan(thrust::hip::par.on(dev_ctx.stream()),
#else
    thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                           offsets_ptr,
                           offsets_ptr + batchs,
                           offsets_ptr);
  }

  GetGpuLaunchConfig1D(dev_ctx, rows, &grid_size, &block_size);
  dim3 grids(grid_size, batchs, 1);
  ConvertCsrCrowsToCooRows<<<grids, block_size>>>(
      csr_crows_data, offsets_ptr, coo_rows_data, batch_ptr, rows);

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_cols_data,
                                            csr_cols_data,
                                            sizeof(int64_t) * non_zero_num,
                                            hipMemcpyDeviceToDevice,
                                            dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_values_data,
                                            csr_values_data,
                                            sizeof(T) * non_zero_num,
                                            hipMemcpyDeviceToDevice,
                                            dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_cols_data,
                                             csr_cols_data,
                                             sizeof(int64_t) * non_zero_num,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_values_data,
                                             csr_values_data,
                                             sizeof(T) * non_zero_num,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));
#endif

  out->SetMember(indices, values, x_dims, true);
}

}  // namespace sparse
}  // namespace pten

PT_REGISTER_KERNEL(dense_to_sparse_coo,
                   GPU,
                   ALL_LAYOUT,
                   pten::sparse::DenseToSparseCooKernel,
                   float,
                   double,
                   pten::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PT_REGISTER_KERNEL(sparse_csr_to_coo,
                   GPU,
                   ALL_LAYOUT,
                   pten::sparse::SparseCsrToCooKernel,
                   float,
                   double,
                   pten::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
