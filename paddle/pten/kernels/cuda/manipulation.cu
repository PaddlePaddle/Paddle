//   Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/api/ext/dispatch.h"
#include "paddle/pten/infermeta/unary.h"
#include "paddle/pten/kernels/cuda/manipulation.h"
#include "paddle/pten/kernels/cuda/utils.h"
#include "paddle/pten/kernels/hybird/cuda/cast_kernel_impl.h"
#include "paddle/pten/kernels/hybird/general/manipulation.h"

namespace pten {

void Reshape(const HIPContext& dev_ctx,
             const DenseTensor& x,
             const ScalarArray& shape,
             DenseTensor* out) {
  auto out_meta = InferMetaFromVecValue(x.meta(), shape.GetData());
  if (x.data() == out->data() && x.numel() == out->numel()) {
    out->Resize(out_meta.dims);
    return;
  }
  pten::Copy(dev_ctx, x, false, out);
  out->Resize(out_meta.dims);
  out->ResetLoD(x.lod());
}

void ReshapeWithXShape(const HIPContext& dev_ctx,
                       const DenseTensor& x,
                       const ScalarArray& shape,
                       DenseTensor* xshape,
                       DenseTensor* out) {
  general::SetXShape(x, xshape);
  Reshape(dev_ctx, x, shape, out);
}

template <typename T>
void Cast(const HIPContext& dev_ctx,
          const DenseTensor& x,
          DataType out_dtype,
          DataType in_dtype,
          DenseTensor* out) {
  PD_VISIT_ALL_TYPES(out_dtype, "CastKernelImpl", ([&] {
                       detail::CastCUDAKernelImpl<T, data_t>(dev_ctx, x, out);
                     }));
}

}  // namespace pten

using float16 = paddle::platform::float16;

#define PTEN_REGISTER_CAST_CUDA_BASE_TYPE(op_name, ...) \
  PT_REGISTER_KERNEL(cast,                              \
                     CUDA,                              \
                     ALL_LAYOUT,                        \
                     pten::Cast,                        \
                     float,                             \
                     double,                            \
                     int,                               \
                     int64_t,                           \
                     int16_t,                           \
                     bool,                              \
                     uint8_t,                           \
                     paddle::platform::float16,         \
                     paddle::platform::complex<float>,  \
                     paddle::platform::complex<double>, \
                     ##__VA_ARGS__) {                   \
    kernel->OutputAt(0).SetDataType(                    \
        paddle::experimental::DataType::UNDEFINED);     \
  }

#if !defined(PADDLE_WITH_HIP)
PTEN_REGISTER_CAST_CUDA_BASE_TYPE(cast, paddle::platform::bfloat16)
#else
PTEN_REGISTER_CAST_CUDA_BASE_TYPE(cast)
#endif

PT_REGISTER_NO_TEMPLATE_KERNEL(reshape, CUDA, ANY, pten::Reshape, ALL_DTYPE) {}
PT_REGISTER_NO_TEMPLATE_KERNEL(
    reshape_with_xshape, CUDA, ANY, pten::ReshapeWithXShape, ALL_DTYPE) {}
