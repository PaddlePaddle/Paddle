//   Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/api/ext/dispatch.h"
#include "paddle/pten/infermeta/unary.h"
#include "paddle/pten/kernels/cuda/manipulation.h"
#include "paddle/pten/kernels/cuda/utils.h"
#include "paddle/pten/kernels/hybird/cuda/cast_kernel_impl.h"
#include "paddle/pten/kernels/hybird/general/manipulation.h"

namespace pten {

template <typename T>
void Flatten(const HIPContext& dev_ctx,
             const DenseTensor& x,
             int start_axis,
             int stop_axis,
             DenseTensor* out) {
  auto out_dims = out->dims();
  pten::Copy(dev_ctx, x, false, out);
  out->Resize(out_dims);
}

// TODO(yuanrisheng): this kernel is for training and xshape is a Intermediate
// Output Tensor，
// is there a more flexible way to deal with this case?
template <typename T>
void FlattenWithXShape(const HIPContext& dev_ctx,
                       const DenseTensor& x,
                       int start_axis,
                       int stop_axis,
                       DenseTensor* out,
                       DenseTensor* xshape) {
  Flatten<T>(dev_ctx, x, start_axis, stop_axis, out);
  general::SetXShape(x, xshape);
}

void ReshapeFromVectorVal(const HIPContext& dev_ctx,
                          const DenseTensor& x,
                          const std::vector<int64_t>& shape,
                          DenseTensor* out) {
  auto out_meta = InferMetaFromVecValue(x.meta(), shape);
  if (x.data() == out->data() && x.numel() == out->numel()) {
    out->Resize(out_meta.dims);
    return;
  }
  pten::Copy(dev_ctx, x, false, out);
  out->Resize(out_meta.dims);
}

void ReshapeFromVectorValWithXShape(const HIPContext& dev_ctx,
                                    const DenseTensor& x,
                                    const std::vector<int64_t>& shape,
                                    DenseTensor* xshape,
                                    DenseTensor* out) {
  general::SetXShape(x, xshape);
  ReshapeFromVectorVal(dev_ctx, x, shape, out);
}

void ReshapeFromDT(const HIPContext& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& shape,
                   DenseTensor* out) {
  auto* shape_data = shape.data<int>();
  auto vector_shape =
      std::vector<int64_t>(shape_data, shape_data + shape.numel());
  ReshapeFromVectorVal(dev_ctx, x, vector_shape, out);
  out->ResetLoD(x.lod());
}

void ReshapeFromDTWithXShape(const HIPContext& dev_ctx,
                             const DenseTensor& x,
                             const DenseTensor& shape,
                             DenseTensor* xshape,
                             DenseTensor* out) {
  general::SetXShape(x, xshape);
  ReshapeFromDT(dev_ctx, x, shape, out);
}

void ReshapeFromVectorDT(const HIPContext& dev_ctx,
                         const DenseTensor& x,
                         const std::vector<DenseTensor>& shape,
                         DenseTensor* out) {
  std::vector<int64_t> vector_shape;
  for (auto& tensor : shape) {
    PADDLE_ENFORCE_EQ(
        tensor.dims(),
        paddle::framework::make_ddim({1}),
        paddle::platform::errors::InvalidArgument(
            "If the element type of 'shape' in ReshapeOp is Tensor, "
            "the element's shape must be [1]. But received the element's shape "
            "is [%s]",
            tensor.dims()));
    vector_shape.push_back(*tensor.data<int32_t>());
  }
  ReshapeFromVectorVal(dev_ctx, x, vector_shape, out);
}

void ReshapeFromVectorDTWithXShape(const HIPContext& dev_ctx,
                                   const DenseTensor& x,
                                   const std::vector<DenseTensor>& shape,
                                   DenseTensor* xshape,
                                   DenseTensor* out) {
  general::SetXShape(x, xshape);
  ReshapeFromVectorDT(dev_ctx, x, shape, out);
}

template <typename T>
void Cast(const HIPContext& dev_ctx,
          const DenseTensor& x,
          DataType out_dtype,
          DataType in_dtype,
          DenseTensor* out) {
  PD_VISIT_ALL_TYPES(out_dtype, "CastKernelImpl", ([&] {
                       detail::CastCUDAKernelImpl<T, data_t>(dev_ctx, x, out);
                     }));
}

}  // namespace pten

using float16 = paddle::platform::float16;

PT_REGISTER_KERNEL(flatten,
                   CUDA,
                   ANY,
                   pten::Flatten,
                   float,
                   float16,
                   double,
                   uint8_t,
                   int8_t,
                   int,
                   int64_t) {}
PT_REGISTER_KERNEL(flatten_mid,
                   CUDA,
                   ANY,
                   pten::FlattenWithXShape,
                   float,
                   double,
                   uint8_t,
                   int8_t,
                   int,
                   int64_t) {}

#define PTEN_REGISTER_CAST_CUDA_BASE_TYPE(op_name, ...) \
  PT_REGISTER_KERNEL(cast,                              \
                     CUDA,                              \
                     ANY,                               \
                     pten::Cast,                        \
                     float,                             \
                     double,                            \
                     int,                               \
                     int64_t,                           \
                     int16_t,                           \
                     bool,                              \
                     uint8_t,                           \
                     paddle::platform::float16,         \
                     paddle::platform::complex<float>,  \
                     paddle::platform::complex<double>, \
                     ##__VA_ARGS__) {                   \
    kernel->OutputAt(0).SetDataType(                    \
        paddle::experimental::DataType::UNDEFINED);     \
  }

#if !defined(PADDLE_WITH_HIP)
PTEN_REGISTER_CAST_CUDA_BASE_TYPE(cast, paddle::platform::bfloat16)
#else
PTEN_REGISTER_CAST_CUDA_BASE_TYPE(cast)
#endif

PT_REGISTER_KERNEL_ALL_DTYPE(reshape, CUDA, ANY, pten::ReshapeFromVectorVal) {}
PT_REGISTER_KERNEL_ALL_DTYPE(reshape_mid,
                             CUDA,
                             ANY,
                             pten::ReshapeFromVectorValWithXShape) {}
PT_REGISTER_KERNEL_ALL_DTYPE(reshape_host, CUDA, ANY, pten::ReshapeFromDT) {
  kernel->InputAt(1).SetBackend(pten::Backend::CPU);
  kernel->InputAt(1).SetDataType(paddle::experimental::DataType::INT32);
}
PT_REGISTER_KERNEL_ALL_DTYPE(reshape_host_mid,
                             CUDA,
                             ANY,
                             pten::ReshapeFromDTWithXShape) {
  kernel->InputAt(1).SetBackend(pten::Backend::CPU);
  kernel->InputAt(1).SetDataType(paddle::experimental::DataType::INT32);
}
PT_REGISTER_KERNEL_ALL_DTYPE(reshape_mulhost,
                             CUDA,
                             ANY,
                             pten::ReshapeFromVectorDT) {
  kernel->InputAt(1).SetBackend(pten::Backend::CPU);
  kernel->InputAt(1).SetDataType(paddle::experimental::DataType::INT32);
}
PT_REGISTER_KERNEL_ALL_DTYPE(reshape_mulhost_mid,
                             CUDA,
                             ANY,
                             pten::ReshapeFromVectorDTWithXShape) {
  kernel->InputAt(1).SetBackend(pten::Backend::CPU);
  kernel->InputAt(1).SetDataType(paddle::experimental::DataType::INT32);
}
