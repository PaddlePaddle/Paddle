/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/pten/kernels/cuda/math.h"

#include "paddle/fluid/operators/reduce_ops/reduce_functor_op.h"
#include "paddle/pten/kernels/hybird/cuda/elementwise/elementwise.h"
#include "paddle/pten/kernels/hybird/cuda/reduce/reduce.h"
#include "paddle/pten/kernels/hybird/eigen/scale.h"
#include "paddle/pten/kernels/hybird/eigen/sign.h"
#include "paddle/pten/kernels/hybird/general/elementwise_functor.h"
#include "paddle/pten/kernels/hybird/general/reduce_impl.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/platform/complex.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/pten/api/lib/utils/tensor_utils.h"
#include "paddle/pten/core/convert_utils.h"
#include "paddle/pten/core/kernel_registry.h"

namespace pten {

/**
 * Util Functors
 */

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n)
      : n_inv(static_cast<T>(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};

/**
 * Kernels
 */

template <typename T>
void Sign(const HIPContext& dev_ctx, const DenseTensor& x, DenseTensor* out) {
  eigen::Sign<HIPContext, T>(dev_ctx, x, out);
}

template <typename T>
void Mean(const HIPContext& dev_ctx,
          const DenseTensor& x,
          const std::vector<int64_t>& dims,
          bool keep_dim,
          bool reduce_all,
          DataType in_dtype,
          DataType out_dtype,
          DenseTensor* out) {
  pten::Reduce<T, paddle::operators::CustomMean>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

template <typename T>
void Scale(const HIPContext& dev_ctx,
           const DenseTensor& x,
           const Scalar& scale,
           float bias,
           bool bias_after_scale,
           DenseTensor* out) {
  eigen::Scale<HIPContext, T>(
      dev_ctx, x, scale.to<float>(), bias, bias_after_scale, out);
}

// Create the definition of ElementwiseAdd
DEFINE_CUDA_ELEMENTWISE_OP(Add)
// Create the definition of ElementwiseSub
DEFINE_CUDA_ELEMENTWISE_OP(Sub)
// Create the definition of ElementwiseMul
DEFINE_CUDA_ELEMENTWISE_OP(Mul)
// Create the definition of ElementwiseDiv
DEFINE_CUDA_ELEMENTWISE_OP(Div)

template <typename T>
void Sum(const HIPContext& dev_ctx,
         const DenseTensor& x,
         const std::vector<int64_t>& dims,
         bool keep_dim,
         bool reduce_all,
         DataType in_dtype,
         DataType out_dtype,
         DenseTensor* out) {
  pten::Reduce<T, paddle::operators::CustomSum>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

}  // namespace pten

using float16 = paddle::platform::float16;
using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_KERNEL(sign, CUDA, ALL_LAYOUT, pten::Sign, float, double, float16) {
}
PT_REGISTER_KERNEL(mean, CUDA, ALL_LAYOUT, pten::Mean, float, double, bool) {}
PT_REGISTER_KERNEL(scale,
                   CUDA,
                   ALL_LAYOUT,
                   pten::Scale,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
PT_REGISTER_KERNEL(add,
                   CUDA,
                   ALL_LAYOUT,
                   pten::ElementwiseAdd,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(subtract,
                   CUDA,
                   ALL_LAYOUT,
                   pten::ElementwiseSub,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(divide,
                   CUDA,
                   ALL_LAYOUT,
                   pten::ElementwiseDiv,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(multiply,
                   CUDA,
                   ALL_LAYOUT,
                   pten::ElementwiseMul,
                   float,
                   double,
                   int,
                   int64_t,
                   bool,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(sum,
                   CUDA,
                   ALL_LAYOUT,
                   pten::Sum,
                   bool,
                   float,
                   double,
                   float16,
                   int,
                   int64_t,
                   complex64,
                   complex128) {
  kernel->OutputAt(0).SetDataType(paddle::experimental::DataType::UNDEFINED);
}
