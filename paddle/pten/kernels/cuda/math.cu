/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/pten/kernels/cuda/math.h"

#include "paddle/pten/kernels/common/eigen/mean.h"
#include "paddle/pten/kernels/common/eigen/scale.h"
#include "paddle/pten/kernels/common/eigen/sign.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/platform/float16.h"
#include "paddle/pten/core/convert_utils.h"
#include "paddle/pten/core/kernel_registry.h"

namespace pten {

/**
 * Util Functors
 */

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n)
      : n_inv(static_cast<T>(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};

/**
 * Kernels
 */

template <typename T>
void Sign(const HIPContext& dev_ctx, const DenseTensor& x, DenseTensor* out) {
  eigen::Sign<HIPContext, T>(dev_ctx, x, out);
}

template <typename T>
void Mean(const HIPContext& dev_ctx, const DenseTensor& x, DenseTensor* out) {
  VLOG(1) << "chenweihang: call new pt mean kernel.";
  // eigen::Mean<HIPContext, T>(dev_ctx, x, out);
  auto size_prob = x.numel();
  const T* x_data = x.data<T>();
  T* out_data = out->mutable_data<T>();
  auto stream = dev_ctx.stream();

  DivideFunctor<T> transformer(size_prob);
  hipcub::TransformInputIterator<T, DivideFunctor<T>, const T*> trans_x(
      x_data, transformer);
  size_t temp_storage_bytes = 0;

  auto err = hipcub::DeviceReduce::Sum(
      nullptr, temp_storage_bytes, trans_x, out_data, size_prob, stream);
  PADDLE_ENFORCE_CUDA_SUCCESS(err);

  pten::DenseTensor tmp(
      TensorMeta(paddle::framework::make_ddim(
                     {static_cast<int64_t>(temp_storage_bytes)}),
                 pten::TransToPtBackend(dev_ctx.GetPlace()),
                 x.data_type(),
                 x.layout()),
      TensorStatus());
  auto* temp_storage = tmp.mutable_data<uint8_t>();
  err = hipcub::DeviceReduce::Sum(
      temp_storage, temp_storage_bytes, trans_x, out_data, size_prob, stream);
  PADDLE_ENFORCE_CUDA_SUCCESS(err);
}

template <typename T>
void Scale(const HIPContext& dev_ctx,
           const DenseTensor& x,
           float scale,
           float bias,
           bool bias_after_scale,
           DenseTensor* out) {
  eigen::Scale<HIPContext, T>(dev_ctx, x, scale, bias, bias_after_scale, out);
}

template <typename T>
void ScaleHost(const HIPContext& dev_ctx,
               const DenseTensor& x,
               const DenseTensor& scale,
               float bias,
               bool bias_after_scale,
               DenseTensor* out) {
  if (paddle::platform::is_gpu_place(scale.place())) {
    throw std::runtime_error("scale host place error.");
  }
  eigen::Scale<HIPContext, T>(dev_ctx,
                               x,
                               static_cast<float>(*scale.data<T>()),
                               bias,
                               bias_after_scale,
                               out);
}

}  // namespace pten

// TODO(chenweihang): replace by better impl
PT_REGISTER_MODULE(MathCUDA);

using float16 = paddle::platform::float16;
PT_REGISTER_KERNEL("sign", CUDA, Any, pten::Sign, float, double, float16) {}
PT_REGISTER_KERNEL("mean", CUDA, Any, pten::Mean, float, double, float16) {}
PT_REGISTER_KERNEL("scale",
                   CUDA,
                   Any,
                   pten::Scale,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
PT_REGISTER_KERNEL("scale.host",
                   CUDA,
                   Any,
                   pten::ScaleHost,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {
  kernel->InputAt(1).SetBackend(pten::Backend::kCPU);
}
