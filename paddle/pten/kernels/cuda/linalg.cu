// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/cuda/linalg.h"

#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/hybird/eigen/dot.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/complex.h"

namespace pten {

template <typename T>
void Dot(const HIPContext& dev_ctx,
         const DenseTensor& x,
         const DenseTensor& y,
         DenseTensor* out) {
  eigen::Dot<HIPContext, T>(dev_ctx, x, y, out);
}

}  // namespace pten

using float16 = paddle::platform::float16;
using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_KERNEL(dot,
                   CUDA,
                   ALL_LAYOUT,
                   pten::Dot,
                   float,
                   double,
                   int,
                   int64_t,
                   complex64,
                   complex128) {}
