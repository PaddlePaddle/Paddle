// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/cuda/linalg.h"

#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/functions/eigen/dot.h"
#include "paddle/pten/kernels/functions/math/matmul_func.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/complex.h"

namespace pten {

template <typename T>
void Dot(const HIPContext& dev_ctx,
         const DenseTensor& x,
         const DenseTensor& y,
         DenseTensor* out) {
  eigen::Dot<HIPContext, T>(dev_ctx, x, y, out);
}

template <typename T>
void Matmul(const HIPContext& dev_ctx,
            const DenseTensor& x,
            const DenseTensor& y,
            bool transpose_x,
            bool transpose_y,
            DenseTensor* out) {
  PADDLE_ENFORCE_NE(paddle::framework::product(x.dims()),
                    0,
                    paddle::platform::errors::InvalidArgument(
                        "The Input(X) dims size must not be equal 0,"
                        " but reviced dims size is 0. "));
  PADDLE_ENFORCE_NE(paddle::framework::product(y.dims()),
                    0,
                    paddle::platform::errors::InvalidArgument(
                        "The Input(Y) dims size must not be equal 0,"
                        " but reviced dims size is 0. "));
  math::MatMulFunction<HIPContext, T>(
      dev_ctx, x, y, out, transpose_x, transpose_y);
}

}  // namespace pten

PT_REGISTER_MODULE(LinalgCUDA);

using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_KERNEL("dot",
                   CUDA,
                   ANY,
                   pten::Dot,
                   float,
                   double,
                   int,
                   int64_t,
                   complex64,
                   complex128) {}

PT_REGISTER_KERNEL("matmul_v2",
                   CUDA,
                   ANY,
                   pten::Matmul,
                   float,
                   double,
                   complex64,
                   complex128) {}
