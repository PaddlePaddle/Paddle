// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/cuda/linalg.h"

#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/functions/eigen/dot.h"
<<<<<<< HEAD
#include "paddle/pten/kernels/functions/math/matmul_func.h"
=======
>>>>>>> b9fdd3bc0f4f22af17a81bb8a50a337b563c876b

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/complex.h"

namespace pten {

template <typename T>
void Dot(const HIPContext& dev_ctx,
         const DenseTensor& x,
         const DenseTensor& y,
         DenseTensor* out) {
  eigen::Dot<HIPContext, T>(dev_ctx, x, y, out);
}

<<<<<<< HEAD
template <typename T>
void Matmul(const HIPContext& dev_ctx,
            const DenseTensor& x,
            const DenseTensor& y,
            bool transpose_x,
            bool transpose_y,
            DenseTensor* out) {
  PADDLE_ENFORCE_NE(paddle::framework::product(x.dims()),
                    0,
                    paddle::platform::errors::InvalidArgument(
                        "The Input(X) dims size must not be equal 0,"
                        " but reviced dims size is 0. "));
  PADDLE_ENFORCE_NE(paddle::framework::product(y.dims()),
                    0,
                    paddle::platform::errors::InvalidArgument(
                        "The Input(Y) dims size must not be equal 0,"
                        " but reviced dims size is 0. "));
  math::MatMulFunction<HIPContext, T>(
      dev_ctx, x, y, out, transpose_x, transpose_y);
}

=======
>>>>>>> b9fdd3bc0f4f22af17a81bb8a50a337b563c876b
}  // namespace pten

PT_REGISTER_MODULE(LinalgCUDA);

<<<<<<< HEAD
using float16 = paddle::platform::float16;
=======
>>>>>>> b9fdd3bc0f4f22af17a81bb8a50a337b563c876b
using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_KERNEL("dot",
                   CUDA,
                   ANY,
                   pten::Dot,
                   float,
                   double,
                   int,
                   int64_t,
                   complex64,
                   complex128) {}

PT_REGISTER_KERNEL("matmul_v2",
                   CUDA,
                   ANY,
                   pten::Matmul,
                   float,
                   double,
                   float16,
                   complex64,
                   complex128) {}
