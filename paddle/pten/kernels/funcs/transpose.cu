#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/ddim.h"
#include "paddle/pten/core/dense_tensor.h"
#include "paddle/pten/kernels/funcs/transpose.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/pten/common/bfloat16.h"
#include "paddle/pten/common/complex.h"
#include "paddle/pten/common/float16.h"

namespace pten {

namespace math {

#define REINTERPRET(T, DST_PTR, SRC_PTR) \
  T* DST_PTR = reinterpret_cast<T*>(SRC_PTR)

template <typename T>
__global__ void TransposeNormalKernel(const T* in_ptr,
                                      T* out_ptr,
                                      int64_t element,
                                      const int64_t* in_stride_ptr,
                                      const int64_t* out_stride_ptr,
                                      const int64_t* axis_ptr,
                                      int rank) {
  CUDA_KERNEL_LOOP(out_idx, element) {
    int64_t in_idx = 0;
    int64_t tmp_idx = out_idx;
    for (int i = 0; i < rank; ++i) {
      const int64_t coordinate = tmp_idx / out_stride_ptr[i];
      tmp_idx -= coordinate * out_stride_ptr[i];
      in_idx += coordinate * in_stride_ptr[axis_ptr[i]];
    }
    out_ptr[out_idx] = in_ptr[in_idx];
  }
}

template <typename T>
struct TransposeNormal<GPUContext, T> {
  // for dims >= 7 situation
  void operator()(const GPUContext& dev_ctx,
                  const pten::DenseTensor& in,
                  pten::DenseTensor* out,
                  const std::vector<int64_t>& axis) {
    const int rank = axis.size();
    auto in_stride = pten::framework::stride(in.dims());
    auto out_stride = pten::framework::stride(out->dims());
    auto* in_ptr = in.data<T>();
    auto* out_ptr = out->mutable_data<T>(dev_ctx.GetPlace());

    // copy in_stride, out_stride, axis to gpu device
    const paddle::platform::CUDAPlace& cuda_place = dev_ctx.GetPlace();
    paddle::platform::CPUPlace cpu_place = paddle::platform::CPUPlace();
    size_t size = 3 * rank * sizeof(int64_t);
    auto cpu_buf_holder = paddle::memory::Alloc(cpu_place, size);
    auto cuda_buf_holder = paddle::memory::Alloc(cuda_place, size);
    REINTERPRET(int64_t, cpu_buf, cpu_buf_holder->ptr());
    REINTERPRET(int64_t, cuda_buf, cuda_buf_holder->ptr());
    for (int i = 0; i < rank; ++i) {
      cpu_buf[i] = in_stride[i];
      cpu_buf[rank + i] = out_stride[i];
      cpu_buf[2 * rank + i] = axis[i];
    }
    paddle::memory::Copy(
        cuda_place, cuda_buf, cpu_place, cpu_buf, size, dev_ctx.stream());
    REINTERPRET(const int64_t, in_stride_ptr, cuda_buf);
    REINTERPRET(const int64_t, out_stride_ptr, cuda_buf + rank);
    REINTERPRET(const int64_t, axis_ptr, cuda_buf + 2 * rank);

    const int MAX_BLOCK_DIM = dev_ctx.GetMaxThreadsPerBlock();
    const int MAX_GRID_DIM =
        dev_ctx.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
    int64_t elements = in.numel();
    int block_size = (elements >= MAX_BLOCK_DIM)
                         ? MAX_BLOCK_DIM
                         : (1 << static_cast<int>(std::log2(elements)));
    int grid_size = elements / block_size;
    grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
    TransposeNormalKernel<T><<<grid_size, block_size, 0, dev_ctx.stream()>>>(
        in_ptr,
        out_ptr,
        elements,
        in_stride_ptr,
        out_stride_ptr,
        axis_ptr,
        rank);
  }
};

// define transpose normal
#define DEFINE_GPU_TRANS_NORMAL(TYPE) \
  template struct TransposeNormal<GPUContext, TYPE>

DEFINE_GPU_TRANS_NORMAL(bool);
DEFINE_GPU_TRANS_NORMAL(int8_t);
DEFINE_GPU_TRANS_NORMAL(uint8_t);
DEFINE_GPU_TRANS_NORMAL(int16_t);
DEFINE_GPU_TRANS_NORMAL(int32_t);
DEFINE_GPU_TRANS_NORMAL(int64_t);
DEFINE_GPU_TRANS_NORMAL(float);
DEFINE_GPU_TRANS_NORMAL(double);
DEFINE_GPU_TRANS_NORMAL(paddle::platform::float16);
DEFINE_GPU_TRANS_NORMAL(paddle::platform::bfloat16);
DEFINE_GPU_TRANS_NORMAL(paddle::platform::complex<float>);
DEFINE_GPU_TRANS_NORMAL(paddle::platform::complex<double>);

}  // namespace math
}  // namespace pten
