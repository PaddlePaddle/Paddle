#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/one_hot_kernel.h"

namespace pten {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename InT, typename OutT>
__global__ void FillOutputKernel(const InT* p_in_data,
                                 OutT* p_out_data,
                                 const int64_t numel,
                                 const int depth) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel && p_in_data[idx] >= 0 && p_in_data[idx] < depth) {
    *(p_out_data + (idx * depth) + p_in_data[idx]) = 1.0;
  }
}

template <typename DeviceContext, typename InT>
struct OneHotV2OpCUDAFunctor {
  const DenseTensor* in_;
  DenseTensor* out_;
  const DeviceContext& ctx_;
  int depth_;

  OneHotV2OpCUDAFunctor(const DenseTensor* in,
                        DenseTensor* out,
                        int depth,
                        const DeviceContext& ctx)
      : in_(in), out_(out), depth_(depth), ctx_(ctx) {}

  template <typename OutT>
  void apply() const {
    auto* p_in_data = in_->data<InT>();
    auto numel = in_->numel();
    auto* p_out_data = out_->mutable_data<OutT>(ctx_.GetPlace());
    auto stream = ctx_.stream();
    paddle::operators::math::set_constant(ctx_, out_, 0.0);

    FillOutputKernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                           PADDLE_CUDA_NUM_THREADS,
                       PADDLE_CUDA_NUM_THREADS,
                       0,
                       stream>>>(p_in_data, p_out_data, numel, depth_);
  }
};

template <typename T, typename Context>
void OneHotKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const Scalar& depth,
                  int dtype,
                  bool allow_out_of_range,
                  DenseTensor* out) {
  int depth_val = depth.to<int>();
  auto out_dims = out->dims();
  if (out_dims[out_dims.size() - 1] == -1) {
    out_dims[out_dims.size() - 1] = depth_val;
    out->Resize(out_dims);
  }
  out->mutable_data<T>(dev_ctx.GetPlace());
  paddle::framework::VisitDataType(
      static_cast<paddle::framework::proto::VarType::Type>(dtype),
      OneHotV2OpCUDAFunctor<Context, T>(&x, out, depth_val, dev_ctx));
}

}  // namespace pten

PT_REGISTER_KERNEL(one_hot, GPU, ALL_LAYOUT, pten::OneHotKernel, int, int64_t) {
}
