#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/cast_kernel.h"

#include "paddle/pten/api/ext/dispatch.h"
#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/kernel_registry.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/aligned_vector.h"
#include "paddle/fluid/platform/bfloat16.h"
#include "paddle/fluid/platform/device/gpu/gpu_helper.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/float16.h"

namespace pten {

template <typename InT, typename OutT, int VecSize>
__global__ void VecCastCUDAKernel(const InT* in, const int64_t N, OutT* out) {
  using LoadT = paddle::platform::AlignedVector<InT, VecSize>;
  using StoreT = paddle::platform::AlignedVector<OutT, VecSize>;

  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int64_t i = idx * VecSize; i < N;
       i += blockDim.x * gridDim.x * VecSize) {
    LoadT in_val;
    paddle::platform::Load<InT, VecSize>(&in[i], &in_val);

    StoreT out_val;
#pragma unroll
    for (int j = 0; j < VecSize; j++) {
      out_val[j] = static_cast<OutT>(in_val[j]);
    }

    paddle::platform::Store<OutT, VecSize>(out_val, &out[i]);
  }
}

template <typename InT, typename OutT>
__global__ void CastCUDAKernel(const InT* in, const int64_t N, OutT* out) {
  CUDA_KERNEL_LOOP(index, N) { out[index] = static_cast<OutT>(in[index]); }
}

template <typename InT, typename OutT>
void CastCUDAKernelImpl(const GPUContext& dev_ctx,
                        const DenseTensor& x,
                        DenseTensor* out) {
  auto* in_data = x.data<InT>();
  auto size = x.numel();
  auto* out_data = out->mutable_data<OutT>();

  paddle::platform::GpuLaunchConfig config =
      paddle::platform::GetGpuLaunchConfig1D(dev_ctx, size);
  int vec_size = paddle::platform::GetVectorizedSize<OutT>(out_data);
  if (!std::is_same<InT, OutT>::value && vec_size == 4 && size % 4 == 0) {
    VecCastCUDAKernel<InT, OutT, 4><<<config.block_per_grid,
                                      config.thread_per_block,
                                      0,
                                      dev_ctx.stream()>>>(
        in_data, size, out_data);
  } else {
    CastCUDAKernel<InT, OutT><<<config.block_per_grid,
                                config.thread_per_block,
                                0,
                                dev_ctx.stream()>>>(in_data, size, out_data);
  }
}

template <typename T, typename ContextT>
void Cast(const ContextT& dev_ctx,
          const DenseTensor& x,
          DataType out_dtype,
          DenseTensor* out) {
  PD_VISIT_ALL_TYPES(out_dtype, "CastCUDAKernelImpl", ([&] {
                       CastCUDAKernelImpl<T, data_t>(dev_ctx, x, out);
                     }));
}

}  // namespace pten

#define PTEN_REGISTER_CAST_CUDA_BASE_TYPE(op_name, ...)     \
  PT_REGISTER_CTX_KERNEL(cast,                              \
                         GPU,                               \
                         ALL_LAYOUT,                        \
                         pten::Cast,                        \
                         float,                             \
                         double,                            \
                         int,                               \
                         int64_t,                           \
                         int16_t,                           \
                         bool,                              \
                         uint8_t,                           \
                         paddle::platform::float16,         \
                         paddle::platform::complex<float>,  \
                         paddle::platform::complex<double>, \
                         ##__VA_ARGS__) {                   \
    kernel->OutputAt(0).SetDataType(                        \
        paddle::experimental::DataType::UNDEFINED);         \
  }

#if !defined(PADDLE_WITH_HIP)
PTEN_REGISTER_CAST_CUDA_BASE_TYPE(cast, paddle::platform::bfloat16)
#else
PTEN_REGISTER_CAST_CUDA_BASE_TYPE(cast)
#endif
