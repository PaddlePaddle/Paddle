/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/pten/common/data_type.h"
#include "paddle/pten/core/convert_utils.h"
#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/gpu/utils.h"

namespace pten {

void Copy(const GPUContext& dev_ctx,
          const DenseTensor& src,
          bool blocking,
          DenseTensor* dst) {
  auto* src_ptr = src.data();
  const auto& src_place = src.place();
  const auto& dst_place = dst->place();

  if (src_place == dst_place && paddle::platform::is_cpu_place(src_place)) {
    PADDLE_THROW(paddle::platform::errors::InvalidArgument(
        "The src and dst tensor are all CPU tensor, you should call copy "
        "function in CPU mode."));
  }

  VLOG(3) << "TensorCopy " << src.dims() << " from " << src.place() << " to "
          << dst_place;

  dst->Resize(src.dims());
  auto* dst_ptr = dst->mutable_data();

  if (src_ptr == dst_ptr && src_place == dst_place) {
    VLOG(3) << "Skip copy the same data async from " << src_place << " to "
            << dst_place;
    return;
  }
  VLOG(4) << "src:" << src_ptr << ", dst:" << dst_ptr;
  CHECK(dst->pten_layout() == src.pten_layout());

  auto size = src.numel() *
              paddle::framework::SizeOfType(TransToProtoVarType(src.dtype()));

  if (paddle::platform::is_cuda_pinned_place(src_place) &&  // NOLINT
      paddle::platform::is_cuda_pinned_place(dst_place)) {
    paddle::memory::Copy(
        BOOST_GET_CONST(paddle::platform::CUDAPinnedPlace, dst_place),
        dst_ptr,
        BOOST_GET_CONST(paddle::platform::CUDAPinnedPlace, src_place),
        src_ptr,
        size);
  } else if (paddle::platform::is_cuda_pinned_place(src_place) &&  // NOLINT
             paddle::platform::is_cpu_place(dst_place)) {
    paddle::memory::Copy(
        BOOST_GET_CONST(paddle::platform::CPUPlace, dst_place),
        dst_ptr,
        BOOST_GET_CONST(paddle::platform::CUDAPinnedPlace, src_place),
        src_ptr,
        size);
  } else if (paddle::platform::is_cpu_place(src_place) &&  // NOLINT
             paddle::platform::is_cuda_pinned_place(dst_place)) {
    paddle::memory::Copy(
        BOOST_GET_CONST(paddle::platform::CUDAPinnedPlace, dst_place),
        dst_ptr,
        BOOST_GET_CONST(paddle::platform::CPUPlace, src_place),
        src_ptr,
        size);
  } else if (paddle::platform::is_gpu_place(src_place) &&  // NOLINT
             paddle::platform::is_cpu_place(dst_place)) {
    auto src_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, src_place);
    auto dst_cpu_place = BOOST_GET_CONST(paddle::platform::CPUPlace, dst_place);
    auto ctx_place = dev_ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        paddle::platform::is_gpu_place(ctx_place),
        true,
        paddle::platform::errors::PreconditionNotMet(
            "Context place error, excepted GPUPlace, but actually %s.",
            ctx_place));
    auto ctx_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(src_gpu_place,
                      ctx_gpu_place,
                      paddle::platform::errors::Unavailable(
                          "Source place and context place do not match, source "
                          "place is %s, context place is %s.",
                          src_gpu_place,
                          ctx_gpu_place));
    auto stream =
        blocking ? nullptr
                 : reinterpret_cast<const paddle::platform::CUDADeviceContext&>(
                       dev_ctx)
                       .stream();
    paddle::memory::Copy(
        dst_cpu_place, dst_ptr, src_gpu_place, src_ptr, size, stream);
  } else if (paddle::platform::is_cpu_place(src_place) &&  // NOLINT
             paddle::platform::is_gpu_place(dst_place)) {
    auto src_cpu_place = BOOST_GET_CONST(paddle::platform::CPUPlace, src_place);
    auto dst_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, dst_place);
    auto ctx_place = dev_ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        paddle::platform::is_gpu_place(ctx_place),
        true,
        paddle::platform::errors::PreconditionNotMet(
            "Context place error, excepted GPUPlace, but actually %s.",
            ctx_place));
    auto ctx_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(dst_gpu_place,
                      ctx_gpu_place,
                      paddle::platform::errors::Unavailable(
                          "Destination place and context place do not match, "
                          "destination place is %s, context place is %s.",
                          dst_gpu_place,
                          ctx_gpu_place));
    auto stream =
        blocking ? nullptr
                 : reinterpret_cast<const paddle::platform::CUDADeviceContext&>(
                       dev_ctx)
                       .stream();
    paddle::memory::Copy(
        dst_gpu_place, dst_ptr, src_cpu_place, src_ptr, size, stream);
  } else if (paddle::platform::is_gpu_place(src_place) &&  // NOLINT
             paddle::platform::is_cuda_pinned_place(dst_place)) {
    auto src_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, src_place);
    auto dst_cuda_pinned_place =
        BOOST_GET_CONST(paddle::platform::CUDAPinnedPlace, dst_place);
    auto ctx_place = dev_ctx.GetPlace();
    PADDLE_ENFORCE_EQ(paddle::platform::is_gpu_place(ctx_place),
                      true,
                      paddle::platform::errors::PreconditionNotMet(
                          "Device context place mismatch. When copying Tensor "
                          "data from GPU memory to CUDA Pinned memory, current "
                          "device context place should be GPU."));
    auto ctx_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(src_gpu_place,
                      ctx_gpu_place,
                      paddle::platform::errors::PreconditionNotMet(
                          "The source GPU device and current device context do "
                          "not match. The source GPU device number is %d, but "
                          "device context GPU number is %d.",
                          src_gpu_place.device,
                          ctx_gpu_place.device));
    auto stream =
        blocking ? nullptr
                 : reinterpret_cast<const paddle::platform::CUDADeviceContext&>(
                       dev_ctx)
                       .stream();
    paddle::memory::Copy(
        dst_cuda_pinned_place, dst_ptr, src_gpu_place, src_ptr, size, stream);
  } else if (paddle::platform::is_cuda_pinned_place(src_place) &&  // NOLINT
             paddle::platform::is_gpu_place(dst_place)) {
    auto src_cuda_pinned_place =
        BOOST_GET_CONST(paddle::platform::CUDAPinnedPlace, src_place);
    auto dst_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, dst_place);
    auto ctx_place = dev_ctx.GetPlace();
    PADDLE_ENFORCE_EQ(paddle::platform::is_gpu_place(ctx_place),
                      true,
                      paddle::platform::errors::PreconditionNotMet(
                          "Device context place mismatch. When copying Tensor "
                          "data from CUDA Pinned memory to GPU memory, current "
                          "device context place should be GPU."));
    auto ctx_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(dst_gpu_place,
                      ctx_gpu_place,
                      paddle::platform::errors::PreconditionNotMet(
                          "The target GPU device and current device context do "
                          "not match. The target GPU device number is %d, but "
                          "device context GPU number is %d.",
                          dst_gpu_place.device,
                          ctx_gpu_place.device));
    auto stream =
        blocking ? nullptr
                 : reinterpret_cast<const paddle::platform::CUDADeviceContext&>(
                       dev_ctx)
                       .stream();
    paddle::memory::Copy(
        dst_gpu_place, dst_ptr, src_cuda_pinned_place, src_ptr, size, stream);
  } else if (paddle::platform::is_gpu_place(src_place) &&  // NOLINT
             paddle::platform::is_gpu_place(dst_place)) {
    auto src_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, src_place);
    auto dst_gpu_place =
        BOOST_GET_CONST(paddle::platform::CUDAPlace, dst_place);
    auto ctx_place = dev_ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        paddle::platform::is_gpu_place(ctx_place),
        true,
        paddle::platform::errors::PreconditionNotMet(
            "Context place error, excepted GPUPlace, but actually %s.",
            ctx_place));
    auto stream =
        blocking ? nullptr
                 : reinterpret_cast<const paddle::platform::CUDADeviceContext&>(
                       dev_ctx)
                       .stream();
    if (paddle::platform::is_same_place(src_place, dst_place)) {
      paddle::memory::Copy(
          dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size, stream);
    } else {
      if (paddle::platform::is_same_place(ctx_place, src_place)) {
        paddle::memory::Copy(
            dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size, stream);
        paddle::platform::DeviceContextPool::Instance()
            .Get(src.place())
            ->Wait();
      } else if (paddle::platform::is_same_place(ctx_place, dst_place)) {
        paddle::platform::DeviceContextPool::Instance()
            .Get(src.place())
            ->Wait();
        paddle::memory::Copy(
            dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size, stream);
      } else {
        PADDLE_THROW(paddle::platform::errors::Unavailable(
            "Context place dose not match the source and destination place."));
      }
    }
  }
}
}  // namespace pten

PT_REGISTER_NO_TEMPLATE_KERNEL(copy, GPU, ALL_LAYOUT, pten::Copy, ALL_DTYPE) {}
