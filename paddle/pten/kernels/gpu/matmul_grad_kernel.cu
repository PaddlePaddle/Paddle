/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/pten/kernels/matmul_grad_kernel.h"

#include "paddle/fluid/platform/complex.h"
#include "paddle/pten/core/kernel_registry.h"

#include "paddle/pten/kernels/impl/matmul_grad_kernel_impl.h"

PT_REGISTER_CTX_KERNEL(matmul_grad,
                       GPU,
                       ALL_LAYOUT,
                       pten::MatmulGradKernel,
                       float,
                       double,
                       paddle::platform::float16,
                       paddle::platform::complex<float>,
                       paddle::platform::complex<double>) {}

PT_REGISTER_CTX_KERNEL(matmul_double_grad,
                       GPU,
                       ALL_LAYOUT,
                       pten::MatmulDoubleGradKernel,
                       float,
                       double,
                       paddle::platform::float16,
                       paddle::platform::complex<float>,
                       paddle::platform::complex<double>) {}

PT_REGISTER_CTX_KERNEL(matmul_triple_grad,
                       GPU,
                       ALL_LAYOUT,
                       pten::MatmulTripleGradKernel,
                       float,
                       double,
                       paddle::platform::float16,
                       paddle::platform::complex<float>,
                       paddle::platform::complex<double>) {}
