// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/split_kernel.h"

#include "paddle/fluid/operators/strided_memcpy.h"
#include "paddle/pten/common/float16.h"
#include "paddle/pten/core/kernel_registry.h"

#include "paddle/pten/kernels/gpu/concat_and_split.h"
namespace pten {

template <typename T, typename Context>
void SplitKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const ScalarArray& num_or_sections,
                 const Scalar& axis_scalar,
                 std::vector<DenseTensor*> outs) {
  // need to infershape output
  if (num_or_sections.FromTensor() || axis_scalar.FromTensor()) {
    std::vector<MetaTensor> out_metas;
    for (size_t i = 0; i < outs.size(); ++i) {
      out_metas.push_back(outs[i]);
    }

    pten::SplitInferMeta(x, num_or_sections, axis_scalar, &out_metas, true);

    for (size_t i = 0; i < out_metas.size(); ++i) {
      outs[i]->Resize(out_metas[i].dims());
    }
  }

  std::vector<const DenseTensor*> shape_refer;
  for (size_t j = 0; j < outs.size(); ++j) {
    dev_ctx.Alloc(outs[j]);
    shape_refer.emplace_back(outs[j]);
  }

  int axis = axis_scalar.to<int>();
  // Sometimes direct copies will be faster, this maybe need deeply analysis.
  if (axis == 0 && outs.size() < 10) {
    paddle::operators::StridedMemcpyWithAxis0<T>(
        dev_ctx, x, shape_refer, &outs);
  } else {
    SplitImpl<T, Context>(dev_ctx, x, shape_refer, axis, &outs);
  }
}

}  // namespace pten

PT_REGISTER_KERNEL(split,
                   GPU,
                   ALL_LAYOUT,
                   pten::SplitKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   bool,
                   pten::dtype::float16,
                   pten::dtype::bfloat16) {}
