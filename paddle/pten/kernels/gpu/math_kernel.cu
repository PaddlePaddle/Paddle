/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/pten/kernels/math_kernel.h"

#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/kernels/funcs/elementwise_functor.h"
#include "paddle/pten/kernels/gpu/elementwise.h"
#include "paddle/pten/kernels/gpu/reduce.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/pten/common/complex.h"
#include "paddle/pten/common/float16.h"
#include "paddle/pten/core/compat/convert_utils.h"
#include "paddle/pten/core/enforce.h"
#include "paddle/pten/core/kernel_registry.h"

namespace pten {

#define DEFINE_CUDA_ELEMENTWISE_OP(name)                             \
  template <typename T, typename Context>                            \
  void name##RawKernel(const Context& dev_ctx,                       \
                       const DenseTensor& x,                         \
                       const DenseTensor& y,                         \
                       int axis,                                     \
                       DenseTensor* out) {                           \
    std::vector<const DenseTensor*> inputs;                          \
    std::vector<DenseTensor*> outputs;                               \
    inputs.emplace_back(&x);                                         \
    inputs.emplace_back(&y);                                         \
    outputs.emplace_back(out);                                       \
    dev_ctx.template Alloc<T>(out);                                  \
    funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(          \
        dev_ctx, inputs, &outputs, axis, funcs::name##Functor<T>()); \
  }

/**
 * Kernels
 */

template <typename T, typename Context>
void MeanRawKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const std::vector<int64_t>& dims,
                   bool keep_dim,
                   bool reduce_all,
                   DenseTensor* out) {
  auto out_dtype = x.dtype();
  pten::Reduce<T, kps::AddFunctor, kps::DivideFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void SumRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const std::vector<int64_t>& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DataType out_dtype,
                  DenseTensor* out) {
  pten::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

// Create the definition of Add
DEFINE_CUDA_ELEMENTWISE_OP(Add)
// Create the definition of Subtract
DEFINE_CUDA_ELEMENTWISE_OP(Subtract)
// Create the definition of Multiply
DEFINE_CUDA_ELEMENTWISE_OP(Multiply)
// Create the definition of Divide
DEFINE_CUDA_ELEMENTWISE_OP(Divide)

}  // namespace pten

using float16 = paddle::platform::float16;
using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_KERNEL(add_raw,
                   GPU,
                   ALL_LAYOUT,
                   pten::AddRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(subtract_raw,
                   GPU,
                   ALL_LAYOUT,
                   pten::SubtractRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(divide_raw,
                   GPU,
                   ALL_LAYOUT,
                   pten::DivideRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(multiply_raw,
                   GPU,
                   ALL_LAYOUT,
                   pten::MultiplyRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   bool,
                   float16,
                   complex64,
                   complex128) {}
PT_REGISTER_KERNEL(sum_raw,
                   GPU,
                   ALL_LAYOUT,
                   pten::SumRawKernel,
                   bool,
                   float,
                   double,
                   float16,
                   int,
                   int64_t,
                   complex64,
                   complex128) {
  kernel->OutputAt(0).SetDataType(paddle::experimental::DataType::UNDEFINED);
}

PT_REGISTER_KERNEL(mean_raw,
                   GPU,
                   ALL_LAYOUT,
                   pten::MeanRawKernel,
                   float,
                   double,
                   bool,
                   float16) {}
