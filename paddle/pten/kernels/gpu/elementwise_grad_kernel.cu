//   Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/elementwise_grad_kernel.h"

#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/copy_kernel.h"
#include "paddle/pten/kernels/funcs/elementwise_functor.h"
#include "paddle/pten/kernels/gpu/elementwise.h"
#include "paddle/pten/kernels/impl/elementwise_grad_kernel_impl.h"

namespace pten {

template <typename T>
void AddGradFunc(const GPUContext& dev_ctx,
                 const DenseTensor& x,
                 const DenseTensor& y,
                 const DenseTensor& out,
                 const DenseTensor& dout,
                 DenseTensor* dx,
                 DenseTensor* dy,
                 int axis = -1) {
  if (dx != nullptr && dy != nullptr && (dx->dims() == dy->dims())) {
    elementwise_add_grad<T>(dev_ctx, x, y, out, dout, dx, dy);
  } else {
    default_elementwise_add_grad<T>(dev_ctx, x, y, out, dout, dx, dy, axis);
  }
}

template <typename T, typename Context>
void AddGradKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   const DenseTensor& dout,
                   int axis,
                   DenseTensor* dx,
                   DenseTensor* dy) {
  pten::AddGradImpl<T>(dev_ctx, x, y, dout, axis, dx, dy, AddGradFunc<T>);
}

template <typename T, typename Context>
void AddDoubleGradKernel(const Context& dev_ctx,
                         const DenseTensor& y,
                         paddle::optional<const DenseTensor&> ddx,
                         paddle::optional<const DenseTensor&> ddy,
                         const DenseTensor& dout,
                         int axis,
                         DenseTensor* ddout) {
  pten::AddDoubleGradImpl<T>(
      dev_ctx,
      y,
      ddx,
      ddy,
      dout,
      axis,
      ddout,
      ElementwiseCompute<funcs::AddFunctor<T>, T>,
      ElementwiseCompute<funcs::InverseAddFunctor<T>, T>);
}

template <typename T, typename Context>
void AddTripleGradKernel(const Context& dev_ctx,
                         const DenseTensor& ddx,
                         const DenseTensor& ddy,
                         const DenseTensor& d_ddout,
                         int axis,
                         DenseTensor* d_ddx,
                         DenseTensor* d_ddy) {
  pten::AddGradImpl<T>(
      dev_ctx, ddx, ddy, d_ddout, axis, d_ddx, d_ddy, AddGradFunc<T>);
}

template <typename T, typename Context>
void SubtractGradKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& y,
                        const DenseTensor& dout,
                        int axis,
                        DenseTensor* dx,
                        DenseTensor* dy) {
  // skip out
  auto* out = &dout;
  if (dx != nullptr && dy != nullptr && (dx->dims() == dy->dims())) {
    elementwise_sub_grad<T>(dev_ctx, x, y, *out, dout, dx, dy);
  } else {
    default_elementwise_sub_grad<T>(dev_ctx, x, y, *out, dout, dx, dy, axis);
  }
}

template <typename T, typename Context>
void SubtractDoubleGradKernel(const Context& dev_ctx,
                              const DenseTensor& y,
                              paddle::optional<const DenseTensor&> ddx,
                              paddle::optional<const DenseTensor&> ddy,
                              const DenseTensor& dout,
                              int axis,
                              DenseTensor* ddout) {
  pten::SubtractDoubleGradImpl<T>(
      dev_ctx,
      y,
      ddx,
      ddy,
      dout,
      axis,
      ddout,
      ElementwiseCompute<funcs::SubtractFunctor<T>, T>);
}

}  // namespace pten

PT_REGISTER_KERNEL(add_grad,
                   GPU,
                   ALL_LAYOUT,
                   pten::AddGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   paddle::platform::float16,
                   paddle::platform::bfloat16,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>) {}

PT_REGISTER_KERNEL(add_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   pten::AddDoubleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   paddle::platform::float16,
                   paddle::platform::bfloat16,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>) {}

PT_REGISTER_KERNEL(add_triple_grad,
                   GPU,
                   ALL_LAYOUT,
                   pten::AddTripleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   paddle::platform::float16,
                   paddle::platform::bfloat16,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>) {}

PT_REGISTER_KERNEL(subtract_grad,
                   GPU,
                   ALL_LAYOUT,
                   pten::SubtractGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   paddle::platform::float16,
                   paddle::platform::bfloat16,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>) {}

PT_REGISTER_KERNEL(subtract_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   pten::SubtractDoubleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   paddle::platform::float16,
                   paddle::platform::bfloat16,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>) {}
