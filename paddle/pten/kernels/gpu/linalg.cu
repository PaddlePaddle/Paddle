// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/gpu/linalg.h"

#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/hybird/eigen/dot.h"
#include "paddle/pten/kernels/hybird/math/matmul_func.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/complex.h"

namespace pten {

template <typename T>
void Dot(const GPUContext& dev_ctx,
         const DenseTensor& x,
         const DenseTensor& y,
         DenseTensor* out) {
  eigen::Dot<GPUContext, T>(dev_ctx, x, y, out);
}

template <typename T>
void Matmul(const GPUContext& dev_ctx,
            const DenseTensor& x,
            const DenseTensor& y,
            bool transpose_x,
            bool transpose_y,
            DenseTensor* out) {
  PADDLE_ENFORCE_NE(paddle::framework::product(x.dims()),
                    0,
                    paddle::platform::errors::InvalidArgument(
                        "The Input(X) dims size must not be equal 0,"
                        " but reviced dims size is 0. "));
  PADDLE_ENFORCE_NE(paddle::framework::product(y.dims()),
                    0,
                    paddle::platform::errors::InvalidArgument(
                        "The Input(Y) dims size must not be equal 0,"
                        " but reviced dims size is 0. "));
  math::MatMulFunction<GPUContext, T>(
      dev_ctx, x, y, out, transpose_x, transpose_y);
}

}  // namespace pten

using float16 = paddle::platform::float16;
using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_KERNEL(dot,
                   GPU,
                   ALL_LAYOUT,
                   pten::Dot,
                   float,
                   double,
                   int,
                   int64_t,
                   complex64,
                   complex128) {}

PT_REGISTER_KERNEL(matmul,
                   GPU,
                   ALL_LAYOUT,
                   pten::Matmul,
                   float,
                   double,
                   float16,
                   complex64,
                   complex128) {}
