// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/dot_kernel.h"

#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/funcs/eigen/common.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/operators/eigen/eigen_function.h"
#include "paddle/fluid/platform/complex.h"

namespace pten {

template <typename T, typename Context>
void DotKernel(const Context& dev_ctx,
               const DenseTensor& x,
               const DenseTensor& y,
               DenseTensor* out) {
  out->mutable_data<T>();
  if (1 == out->dims().size()) {
    auto eigen_out = pten::EigenScalar<T>::From(*out);
    auto eigen_x = pten::EigenVector<T>::Flatten(x);
    auto eigen_y = pten::EigenVector<T>::Flatten(y);

    auto& dev = *dev_ctx.eigen_device();
    eigen_out.device(dev) = (eigen_x * eigen_y).sum();
  } else {
    auto eigen_out = pten::EigenMatrix<T>::From(*out);
    auto eigen_x = pten::EigenMatrix<T>::From(x);
    auto eigen_y = pten::EigenMatrix<T>::From(y);

    auto& dev = *dev_ctx.eigen_device();
    eigen_out.device(dev) = (eigen_x * eigen_y).sum(Eigen::DSizes<int, 1>(1));
  }
}

}  // namespace pten

using complex64 = ::paddle::platform::complex<float>;
using complex128 = ::paddle::platform::complex<double>;

PT_REGISTER_CTX_KERNEL(dot,
                       GPU,
                       ALL_LAYOUT,
                       pten::DotKernel,
                       float,
                       double,
                       int,
                       int64_t,
                       complex64,
                       complex128) {}
