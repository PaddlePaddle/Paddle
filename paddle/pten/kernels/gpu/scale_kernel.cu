/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/pten/kernels/scale_kernel.h"

#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/kernel_registry.h"
#include "paddle/pten/kernels/funcs/elementwise_base.h"
// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/float16.h"

namespace pten {

template <typename InT>
struct ScaleFunctor {
  InT bias;
  InT scale;
  bool bias_after_scale;

  ScaleFunctor(InT scale_data, InT bias_data, bool is_bias_after_sacle) {
    scale = scale_data;
    bias = bias_data;
    bias_after_scale = is_bias_after_sacle;
  }

  __device__ __forceinline__ InT operator()(const InT x) const {
    if (bias_after_scale) {
      return scale * x + bias;
    } else {
      return scale * (x + bias);
    }
  }
};

template <typename T, typename Context>
void ScaleKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const Scalar& scale,
                 float bias,
                 bool bias_after_scale,
                 DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  std::vector<DenseTensor*> outputs;
  inputs.emplace_back(&x);
  outputs.emplace_back(out);
  out->mutable_data<T>();
  pten::funcs::LaunchSameDimsElementwiseCudaKernel<ElementwiseType::kUnary,
                                                   T,
                                                   T>(
      dev_ctx,
      inputs,
      &outputs,
      ScaleFunctor<T>(scale.to<T>(), static_cast<T>(bias), bias_after_scale));
}

}  // namespace pten

PT_REGISTER_KERNEL(scale,
                   GPU,
                   ALL_LAYOUT,
                   pten::ScaleKernel,
                   float,
                   double,
                   paddle::platform::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
