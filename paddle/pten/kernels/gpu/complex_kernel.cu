// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/pten/kernels/complex_kernel.h"
#include "paddle/pten/kernels/impl/complex_kernel_impl.h"

#include "paddle/pten/backends/gpu/gpu_context.h"
#include "paddle/pten/core/kernel_registry.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/complex.h"

PT_REGISTER_KERNEL(conj,
                   GPU,
                   ALL_LAYOUT,
                   pten::ConjKernel,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>,
                   float,
                   double,
                   int,
                   int64_t) {}
