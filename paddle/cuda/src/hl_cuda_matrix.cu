#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */


#include "hl_base.h"
#include "hl_matrix.h"
#include "hl_matrix_ops.cuh"
#include "hl_matrix_apply.cuh"
#include "hl_sequence.h"
#include "hl_sparse.ph"
#include "paddle/utils/Logging.h"
#include "hl_device_functions.cuh"
#include "hl_gpu_matrix_kernel.cuh"

DEFINE_MATRIX_UNARY_OP(Zero, a = 0);
DEFINE_MATRIX_TERNARY_PARAMETER_OP(_add, TWO_PARAMETER, c = p1*a + p2*b);
void hl_matrix_add(real *A_d,
                   real *B_d,
                   real *C_d,
                   int dimM,
                   int dimN,
                   real alpha,
                   real beta) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(B_d);
  CHECK_NOTNULL(C_d);

  hl_gpu_apply_ternary_op
    <real, ternary::_add<real>, 0, 0>(ternary::_add<real>(alpha, beta),
                                      A_d,
                                      B_d,
                                      C_d,
                                      dimM,
                                      dimN,
                                      dimN,
                                      dimN,
                                      dimN);
  CHECK_SYNC("hl_matrix_add failed");
}

#ifdef PADDLE_TYPE_DOUBLE
    #define THRESHOLD   128
#else
    #define THRESHOLD   64
#endif
__device__ __forceinline__
void findMax(real* I,
             real* dfMax_s,
             int blockSize,
             int base,
             int curIdx,
             int nextIdx,
             int dimN,
             real* max) {
  dfMax_s[base] = -1.0e20;
  while (curIdx < dimN) {
    if (dfMax_s[base] < I[nextIdx]) {
      dfMax_s[base] = I[nextIdx];
    }
    nextIdx += blockSize;
    curIdx += blockSize;
  }
  __syncthreads();

  for (int stride = blockSize >> 1; stride > 0; stride >>= 1) {
    __syncthreads();
    if (base < stride) {
      nextIdx = base + stride;
      if (dfMax_s[base] < dfMax_s[nextIdx]) {
          dfMax_s[base] = dfMax_s[nextIdx];
      }
    }
  }

  if (0 == base)  {
    max[0] = dfMax_s[0];
  }
  __syncthreads();
}

__device__ __forceinline__
void subMaxAndExp(real* I,
                  real* O,
                  int curIdx,
                  int nextIdx,
                  int blockSize,
                  int dimN,
                  real max) {
  real val;
  while (curIdx < dimN) {
    val = I[nextIdx] - max;
    if (val < -THRESHOLD) {
      val = -THRESHOLD;
    }
    I[nextIdx] = val;
#ifndef PADDLE_TYPE_DOUBLE
    O[nextIdx] = __expf(val);
#else
    O[nextIdx] = exp(val);
#endif
    nextIdx += blockSize;
    curIdx += blockSize;
  }
  __syncthreads();
}

__device__ __forceinline__
void valueSum(real* O,
              real* dfMax_s,
              int blockSize,
              int base,
              int curIdx,
              int nextIdx,
              int dimN) {
  dfMax_s[base] = 0;
  while (curIdx < dimN) {
    dfMax_s[base] += O[nextIdx];
    nextIdx += blockSize;
    curIdx += blockSize;
  }
  __syncthreads();

  for (int stride = blockSize >> 1; stride > 0; stride >>= 1) {
    __syncthreads();
    if (base < stride) {
      nextIdx = base + stride;
      dfMax_s[base] += dfMax_s[nextIdx];
    }
  }
  __syncthreads();
}

__device__ __forceinline__
void divSum(real* O,
            real sum,
            int curIdx,
            int nextIdx,
            int blockSize,
            int dimN) {
  while (curIdx < dimN) {
    O[nextIdx] /= sum;
    nextIdx += blockSize;
    curIdx += blockSize;
  }
}

__device__ __forceinline__
void softmax(real* I,
             real* O,
             real* dfMax_s,
             int blockSize,
             int base,
             int curIdx,
             int nextIdx,
             int dimN) {
  __shared__ real max;

  // find the max number
  findMax(I, dfMax_s, blockSize, base, curIdx,
          nextIdx, dimN, &max);

  // sub max Value and do Exp operation
  subMaxAndExp(I, O, base, nextIdx, blockSize, dimN, max);

  // add dimN values into blockDim.x buffer
  // sum is in dfMax_s[0]
  valueSum(O, dfMax_s, blockSize, base, curIdx, nextIdx, dimN);

  // divided by sum
  divSum(O, dfMax_s[0], curIdx, nextIdx, blockSize, dimN);
}

template<int blockSize>
__global__ void KeMatrixSoftMax(real *O, real *I, int dimN) {
  int base = threadIdx.x;
  __shared__ real dfMax_s[blockSize];
  int nextIdx = blockIdx.x * dimN + base;
  int curIdx = base;

  softmax(I, O, dfMax_s, blockSize, base, curIdx, nextIdx, dimN);
}

void hl_matrix_softmax(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  dim3 block(512, 1);
  dim3 grid(dimM, 1);
  KeMatrixSoftMax<512>
           <<<grid, block, 0, STREAM_DEFAULT>>>(C_d, A_d, dimN);
  CHECK_SYNC("hl_matrix_softmax failed");
}

template<int blockSize>
__global__ void KeSequenceSoftMax(real *O, real *I, const int* index) {
  int base = threadIdx.x;
  int bid = blockIdx.x;
  __shared__ real dfMax_s[blockSize];

  int start = index[bid];
  int dimN = index[bid + 1] - start;

  int nextIdx = start + base;
  int curIdx = base;

  softmax(I, O, dfMax_s, blockSize, base, curIdx, nextIdx, dimN);
}

void hl_sequence_softmax_forward(real *A_d,
                                 real *C_d,
                                 const int* index,
                                 int numSequence) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  dim3 block(512, 1);
  dim3 grid(numSequence, 1);
  KeSequenceSoftMax<512>
           <<<grid, block, 0, STREAM_DEFAULT>>>(C_d, A_d, index);
  CHECK_SYNC("hl_sequence_softmax_forward failed");
}

__global__ void KeMatrixDerivative(real *grad_d,
                                   real *output_d,
                                   real *sftmaxSum_d,
                                   int dimM,
                                   int dimN) {
  int rowIdx = blockIdx.x*blockDim.x + threadIdx.x;
  int colIdx = blockIdx.y*blockDim.y + threadIdx.y;
  int index;

  if (rowIdx < dimM && colIdx < dimN) {
    index = rowIdx*dimN + colIdx;
    grad_d[index] = output_d[index] * (grad_d[index] - sftmaxSum_d[rowIdx]);
  }
}

void hl_matrix_softmax_derivative(real *grad_d,
                                  real *output_d,
                                  real *sftmaxSum_d,
                                  int dimM,
                                  int dimN) {
  CHECK_NOTNULL(grad_d);
  CHECK_NOTNULL(output_d);
  CHECK_NOTNULL(sftmaxSum_d);

  int blocksX = (dimM + 0) / 1;
  int blocksY = (dimN + 1024 -1) / 1024;
  dim3 threads(1, 1024);
  dim3 grid(blocksX, blocksY);

  KeMatrixDerivative<<< grid, threads, 0, STREAM_DEFAULT >>>
           (grad_d, output_d, sftmaxSum_d, dimM, dimN);
  CHECK_SYNC("hl_matrix_softmax_derivative failed");
}

template<int blockSize>
__global__ void KeMatrixClassificationError(real* in_A,
                                            int* in_B,
                                            real* out_C,
                                            int dimN) {
  __shared__ real max_s[blockSize];
  __shared__ int max_l[blockSize];
  const int tid = threadIdx.x;
  const int rowId = blockIdx.x;

  max_s[tid] = -1e30f;
  in_A += rowId * dimN;
  real tmp;
  for (int colId = tid; colId < dimN; colId += blockSize) {
    tmp = in_A[colId];
    if (max_s[tid] < tmp) {
      max_s[tid] = tmp;
      max_l[tid] = colId;
    }
  }
  __syncthreads();

  for (int stride = blockSize/2; stride > 0; stride = stride/2) {
    if (tid < stride) {
      if (max_s[tid] < max_s[tid + stride]) {
        max_s[tid] = max_s[tid + stride];
        max_l[tid] = max_l[tid + stride];
      }
    }
    __syncthreads();
  }
  __syncthreads();

  if (tid == 0) {
    out_C[rowId] = (max_l[0] == in_B[rowId] ? 0 : 1.0f);
  }
}

void hl_matrix_classification_error(real* A_d,
                                    int* B_d,
                                    real* C_d,
                                    int dimM,
                                    int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(B_d);
  CHECK_NOTNULL(C_d);

  // each sample is calculated by one block
  KeMatrixClassificationError<1024><<< dimM, 1024, 0, STREAM_DEFAULT >>>
    (A_d, B_d, C_d, dimN);
  CHECK_SYNC("hl_matrix_classification_error");
}

__global__ void KeMatrixMultiBinaryCrossEntropy(real* output,
                                                real* entropy,
                                                int* row,
                                                int* col,
                                                int dimM,
                                                int dimN) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < dimM) {
    for (int i = 0; i < dimN; i ++) {
      entropy[index] -= log(1 - output[index * dimN + i]);
    }
    int *row_col = col + row[index];
    int col_num = row[index + 1] - row[index];
    for (int i = 0; i < col_num; i ++) {
      real o = output[index * dimN + row_col[i]];
      entropy[index] -= log(o / (1 - o));
    }
  }
}

void hl_matrix_multi_binary_cross_entropy(real* output,
                                          real* entropy,
                                          hl_sparse_matrix_s csr_mat,
                                          int dimM,
                                          int dimN) {
  CHECK_NOTNULL(output);
  CHECK_NOTNULL(entropy);
  CHECK_NOTNULL(csr_mat);
  CHECK_EQ(csr_mat->format, HL_SPARSE_CSR);
  int n_threads = 1024;
  int blocks = (dimM + n_threads - 1) / n_threads;
  dim3 threads(n_threads);
  dim3 grid(blocks);
  hl_csr_matrix mat = (hl_csr_matrix)(csr_mat->matrix);
  KeMatrixMultiBinaryCrossEntropy<<< grid, threads, 0, STREAM_DEFAULT >>>
          (output, entropy, mat->csr_row, mat->csr_col, dimM, dimN);
  CHECK_SYNC("hl_matrix_multi_binary_cross_entropy failed");
}

__global__ void KeMatrixMultiBinaryCrossEntropyBp(real* output,
                                                  real* grad,
                                                  int* row,
                                                  int* col,
                                                  int dimM,
                                                  int dimN) {
  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (row_idx < dimM) {
    for (int i = 0; i < dimN; i ++) {
      int index = row_idx * dimN + i;
      grad[index] += 1.0 / (1 - output[index]);
    }
    int col_num = row[row_idx + 1] - row[row_idx];
    int *row_col = col + row[row_idx];
    for (int i = 0; i < col_num; i ++) {
      int index = row_idx * dimN + row_col[i];
      grad[index] -= 1.0 / (output[index] * (1 - output[index]));
    }
  }
}

void hl_matrix_multi_binary_cross_entropy_bp(real* output,
                                             real* grad,
                                             hl_sparse_matrix_s csr_mat,
                                             int dimM,
                                             int dimN) {
  CHECK_NOTNULL(output);
  CHECK_NOTNULL(grad);
  CHECK_NOTNULL(csr_mat);
  CHECK_EQ(csr_mat->format, HL_SPARSE_CSR);
  int n_threads = 1024;
  int blocks = (dimM + n_threads - 1) / n_threads;
  dim3 threads(n_threads);
  dim3 grid(blocks);
  hl_csr_matrix mat = (hl_csr_matrix)(csr_mat->matrix);
  KeMatrixMultiBinaryCrossEntropyBp<<< grid, threads, 0, STREAM_DEFAULT >>>
          (output, grad, mat->csr_row, mat->csr_col, dimM, dimN);
  CHECK_SYNC("hl_matrix_multi_binary_cross_entropy_bp failed");
}

__global__ void KeMatrixCrossEntropy(real* O,
                                     real* E,
                                     int* label,
                                     int dimM,
                                     int dimN) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int newBase;
  if (index < dimM) {
    newBase = label[index];
    newBase = newBase % dimN;
    E[index] = -log(O[index * dimN + newBase]);
  }
}

void hl_matrix_cross_entropy(real* A_d,
                             real* C_d,
                             int* label_d,
                             int dimM,
                             int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  int blocks = (dimM + 1024 - 1) / 1024;
  dim3 threads(1024, 1);
  dim3 grid(blocks, 1);
  KeMatrixCrossEntropy<<< grid, threads, 0, STREAM_DEFAULT >>>
           (A_d, C_d, label_d, dimM, dimN);
  CHECK_SYNC("hl_matrix_cross_entropy failed");
}

__global__ void KeMatrixCrossEntropyBp(real* grad_d,
                                       real* output_d,
                                       int* label_d,
                                       int dimM,
                                       int dimN) {
  int rowIdx = blockIdx.x*blockDim.x + threadIdx.x;
  int colIdx = blockIdx.y*blockDim.y + threadIdx.y;
  int index;
  if (rowIdx < dimM && colIdx < dimN) {
    index = rowIdx*dimN + colIdx;
    if (label_d[rowIdx] == colIdx) {
      grad_d[index] -= 1.0f / output_d[index];
    }
  }
}

void hl_matrix_cross_entropy_bp(real* grad_d,
                                real* output_d,
                                int* label_d,
                                int dimM,
                                int dimN) {
  CHECK_NOTNULL(grad_d);
  CHECK_NOTNULL(output_d);
  CHECK_NOTNULL(label_d);

  int blocksX = (dimM + 0)/1;
  int blocksY = (dimN + 1024 -1) / 1024;
  dim3 threads(1, 1024);
  dim3 grid(blocksX, blocksY);
  KeMatrixCrossEntropyBp<<< grid, threads, 0, STREAM_DEFAULT >>>
           (grad_d, output_d, label_d, dimM, dimN);
  CHECK_SYNC("hl_matrix_cross_entropy_bp failed");
}

void hl_matrix_zero_mem(real* data, int num) {
  hl_gpu_apply_unary_op(
        unary::Zero<real>(), data, 1, num, num);
}

__global__ void KeParamReluForward(real* output,
                                   real* input,
                                   real* w,
                                   int width,
                                   int height,
                                   int partial_sum) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  if (tx < width && ty < height) {
    int index = ty * width + tx;
    output[index] = input[index] > 0 ? input[index] :
        input[index] * w[tx / partial_sum];
  }
}

void hl_param_relu_forward(real* output,
                           real* input,
                           real* w,
                           int width,
                           int height,
                           int partial_sum) {
  CHECK_NOTNULL(output);
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(w);
  dim3 threads(16, 16);
  int blockX = (width + 16 - 1) / 16;
  int blockY = (height + 16 -1) / 16;
  dim3 grid(blockX, blockY);
  KeParamReluForward<<<grid, threads, 0, STREAM_DEFAULT>>>
    (output, input, w, width, height, partial_sum);
  CHECK_SYNC("hl_param_relu_forward failed");
}

template<int blockSize>
__global__ void KeParamReluBackWardW(real* grad_w,
                                     real* grad_o,
                                     real* input,
                                     int width,
                                     int height,
                                     int partial_sum) {
  const int tid = threadIdx.x;
  __shared__ real temp[blockSize];
  grad_o += partial_sum * blockIdx.x;
  input += partial_sum * blockIdx.x;
  real tmp = 0.0;
  for (int index = tid; index < partial_sum * height; index += blockSize) {
    int row = index / partial_sum;
    int offset = row * width + (index - row * partial_sum);
    if (input[offset] < 0) {
      tmp += grad_o[offset] * input[offset];
    }
  }
  temp[tid] = tmp;
  __syncthreads();
  for (int s = blockSize / 2; s > 0; s >>= 1) {
    if (tid < s) {
      temp[tid] += temp[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) {
    grad_w[blockIdx.x] += temp[0];
  }
}

void hl_param_relu_backward_w(real* grad_w,
                              real* grad_o,
                              real* input,
                              int width,
                              int height,
                              int partial_sum) {
  CHECK_NOTNULL(grad_w);
  CHECK_NOTNULL(grad_o);
  CHECK_NOTNULL(input);
  const int blockSize = 1024;
  int grid_num = width / partial_sum;
  dim3 threads(blockSize, 1);
  dim3 grid(grid_num, 1);
  KeParamReluBackWardW<blockSize><<<grid, threads, 0, STREAM_DEFAULT>>>
    (grad_w, grad_o, input, width, height, partial_sum);
  CHECK_SYNC("hl_param_relu_backward_w failed");
}

__global__ void KeParamReluBackwardDiff(real* grad_o,
                                        real* input,
                                        real* w,
                                        real* diff,
                                        int width,
                                        int height,
                                        int partial_sum) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  if (tx < width && ty < height) {
    int index = ty * width + tx;
    diff[index] += grad_o[index] * (input[index] > 0 ? 1 : w[tx / partial_sum]);
  }
}

void hl_param_relu_backward_diff(real* grad_o,
                                 real* data,
                                 real* w,
                                 real* diff,
                                 int width,
                                 int height,
                                 int partial_sum) {
  CHECK_NOTNULL(grad_o);
  CHECK_NOTNULL(data);
  CHECK_NOTNULL(w);
  CHECK_NOTNULL(diff);
  dim3 threads(16, 16);
  int blockX = (width + 16 - 1) / 16;
  int blockY = (height + 16 -1) / 16;
  dim3 grid(blockX, blockY);
  KeParamReluBackwardDiff<<<grid, threads, 0, STREAM_DEFAULT>>>
      (grad_o, data, w, diff, width, height, partial_sum);
  CHECK_SYNC("hl_param_relu_backward_diff failed");
}

template<int blockSize>
__global__ void KeCosSim(real* output,
                         real* input1,
                         real* input2,
                         int width,
                         int input1_height,
                         int input2_height,
                         real scale) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  __shared__ real xx[blockSize];
  __shared__ real yy[blockSize];
  __shared__ real xy[blockSize];

  xx[tid] = 0.0;
  yy[tid] = 0.0;
  xy[tid] = 0.0;
  __syncthreads();

  input1 += ty * width;
  if (input2_height > 1) {
    input2 += ty * width;
  }
  for (int index = tid; index < width; index += blockSize) {
    real x = input1[index];
    real y = input2[index];
    xx[tid] += x * x;
    yy[tid] += y * y;
    xy[tid] += x * y;
  }
  __syncthreads();

  for (int s = blockSize / 2; s > 0; s >>= 1) {
    if (tid < s) {
      xx[tid] += xx[tid + s];
      yy[tid] += yy[tid + s];
      xy[tid] += xy[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) {
    output[ty] = scale * xy[0] / (sqrt(xx[0]) * sqrt(yy[0]));
  }
}

void hl_cossim(real* output,
               real* input1,
               real* input2,
               int width,
               int input1_height,
               int input2_height,
               real scale) {
  CHECK_NOTNULL(output);
  CHECK_NOTNULL(input1);
  CHECK_NOTNULL(input2);
  const int blockSize = 256;
  dim3 threads(blockSize, 1);
  dim3 grid(1, input1_height);

  KeCosSim<blockSize><<<grid, threads, 0, STREAM_DEFAULT>>>
    (output, input1, input2, width, input1_height, input2_height, scale);
  CHECK_SYNC("hl_cossim failed");
}

template<int blockSize>
__global__ void KeCosSimDerivative(real* grad,
                                   real* output,
                                   real* prevOutX,
                                   real* prevOutY,
                                   real* prevGradX,
                                   real* prevGradY,
                                   int width,
                                   int input1_height,
                                   int input2_height,
                                   real scale) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  __shared__ real xx[blockSize];
  __shared__ real yy[blockSize];
  __shared__ real xy[blockSize];

  xx[tid] = 0.0;
  yy[tid] = 0.0;
  xy[tid] = 0.0;
  __syncthreads();

  prevOutX += ty * width;
  prevGradX += ty * width;
  if (input2_height > 1) {
    prevOutY += ty * width;
    prevGradY += ty * width;
  }
  for (int index = tid; index < width; index += blockSize) {
    real x = prevOutX[index];
    real y = prevOutY[index];
    xx[tid] += x * x;
    yy[tid] += y * y;
    xy[tid] += x * y;
  }
  __syncthreads();

  for (int s = blockSize / 2; s > 0; s >>= 1) {
    if (tid < s) {
      xx[tid] += xx[tid + s];
      yy[tid] += yy[tid + s];
      xy[tid] += xy[tid + s];
    }
    __syncthreads();
  }
  if (xy[0] == 0) {
    real reciprocal = 1.0 / (sqrt(xx[0]) * sqrt(yy[0]));
    for (int index = tid; index < width; index += blockSize) {
      prevGradX[index] +=
        scale * grad[ty] * prevOutY[index] * reciprocal;
      if (input2_height > 1) {
        prevGradY[index] +=
          scale * grad[ty] * prevOutX[index] * reciprocal;
      } else {
        paddle::paddleAtomicAdd(prevGradY + index,
          scale * grad[ty] * prevOutX[index] * reciprocal);
      }
    }
  } else {
    real reciprocalXY = 1.0 / xy[0];
    real reciprocalSquareSumX = 1.0 / xx[0];
    real reciprocalSquareSumY = 1.0 / yy[0];
    for (int index = tid; index < width; index += blockSize) {
      prevGradX[index] += output[ty] * grad[ty] *
        (prevOutY[index] * reciprocalXY -
         prevOutX[index] * reciprocalSquareSumX);
      if (input2_height > 1) {
        prevGradY[index] += output[ty] * grad[ty] *
          (prevOutX[index] * reciprocalXY -
           prevOutY[index] * reciprocalSquareSumY);
      } else {
        paddle::paddleAtomicAdd(prevGradY + index, output[ty] * grad[ty] *
          (prevOutX[index] * reciprocalXY -
           prevOutY[index] * reciprocalSquareSumY));
      }
    }
  }
}


void hl_cossim_derivative(real* grad,
                          real* output,
                          real* prevOutX,
                          real* prevOutY,
                          real* prevGradX,
                          real* prevGradY,
                          int width,
                          int input1_height,
                          int input2_height,
                          real scale) {
  CHECK_NOTNULL(grad);
  CHECK_NOTNULL(output);
  CHECK_NOTNULL(prevOutX);
  CHECK_NOTNULL(prevOutY);
  CHECK_NOTNULL(prevGradX);
  CHECK_NOTNULL(prevGradY);
  const int blockSize = 256;
  dim3 threads(blockSize, 1);
  dim3 grid(1, input1_height);
  KeCosSimDerivative<blockSize><<<grid, threads, 0, STREAM_DEFAULT>>>
    (grad, output, prevOutX, prevOutY, prevGradX, prevGradY, width,
        input1_height, input2_height, scale);
  CHECK_SYNC("hl_cossim_derivate failed");
}

__global__ void KeMatrixAddSharedBias(real* A,
                                      real* B,
                                      const int channel,
                                      const int M,
                                      const int N,
                                      real scale) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int dim = N / channel;
  if (index < M * N) {
    int i = index % N;
    i = i / dim;
    A[index] += scale * B[i];
  }
}

void hl_matrix_add_shared_bias(real* A_d,
                               real* B_d,
                               const int channel,
                               const int dimM,
                               const int dimN,
                               real scale) {
  const int blocks = 512;
  const int grids = DIVUP(dimM * dimN, blocks);
  KeMatrixAddSharedBias<<<grids, blocks, 0, STREAM_DEFAULT>>>
    (A_d, B_d, channel, dimM, dimN, scale);
  CHECK_SYNC("hl_matrix_add_shared_bias failed");
}


template <int blockSize>
__global__ void KeMatrixCollectSharedBias(real *B,
                                          real *A,
                                          const int channel,
                                          const int M,
                                          const int N,
                                          const int dim,
                                          const int limit,
                                          real scale) {
  if (dim < limit) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < channel) {
      real sum = 0.0;
      for (int i = 0; i < M; ++i) {
        for (int j = 0; j < dim; ++j) {
          sum += A[i * N + index * dim + j];
        }
      }
      B[index] += scale * sum;
    }
  } else {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ real smem[blockSize];
    real sum = 0.0;
    for (int j = 0; j < ((dim * M + blockSize - 1) / blockSize); ++j) {
      int n = j * blockSize + tid;
      int m = n / dim;
      int w = n % dim;
      smem[tid] =  (m < M && w < dim) ? A[m * N + bid * dim + w] : 0.0;
      __syncthreads();
      simpleReduce(smem, tid, blockSize);
      sum += smem[0];
    }
    if (tid == 0) {
      B[bid] += scale * sum;
    }
  }
}

void hl_matrix_collect_shared_bias(real* B_d,
                                   real* A_d,
                                   const int channel,
                                   const int dimM,
                                   const int dimN,
                                   real scale) {
  const int dim = dimN / channel;
  const int blocks = 256;
  const int limit = 64;
  int grids = (dimM * dim) < limit ? DIVUP(channel, blocks) : channel;

  KeMatrixCollectSharedBias<blocks>
      <<< grids, blocks, 0, STREAM_DEFAULT>>>
      (B_d, A_d, channel, dimM, dimN, dim, limit, scale);
  CHECK_SYNC("hl_matrix_collect_shared_bias failed");
}
