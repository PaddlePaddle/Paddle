#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */


#include "hl_base.h"
#include "hl_cuda.h"
#include "hl_cuda.ph"
#include "hl_aggregate.h"
#include "hl_thread.ph"
#include "hl_matrix_base.cuh"
#include "paddle/utils/Logging.h"

/**
 * @brief   matrix row operator.
 */
template<class Agg, int blockSize>
__global__ void KeMatrixRowOp(Agg agg,
                              real *E,
                              real *Sum,
                              int dimN) {
  __shared__ real sum_s[blockSize];
  int cnt = (dimN + blockSize -1) / blockSize;
  int rowId = blockIdx.x + blockIdx.y*gridDim.x;
  int index = rowId*dimN;
  int tid = threadIdx.x;
  int lmt = tid;

  real tmp = agg.init();
  for (int ii = 0; ii < cnt && lmt < dimN; ii++) {
    tmp = agg(tmp, E[index + lmt]);
    lmt += blockSize;
  }
  sum_s[tid] = tmp;
  __syncthreads();

  for (int stride = blockSize/2; stride > 0; stride = stride/2) {
    if (tid < stride) {
      sum_s[tid] = agg(sum_s[tid], sum_s[tid + stride]);
    }
    __syncthreads();
  }
  __syncthreads();

  if (tid == 0) {
    Sum[rowId] = sum_s[0];
  }
}

template <class Agg>
void hl_matrix_row_op(Agg agg,
                      real *A_d,
                      real *C_d,
                      int dimM,
                      int dimN) {
  int blocksX = dimM;
  int blocksY = 1;
  dim3 threads(128, 1);
  dim3 grid(blocksX, blocksY);

  KeMatrixRowOp<Agg, 128><<< grid, threads, 0, STREAM_DEFAULT >>>
           (agg, A_d, C_d, dimN);
}

void hl_matrix_row_sum(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  hl_matrix_row_op(aggregate::sum(),
                   A_d,
                   C_d,
                   dimM,
                   dimN);
  CHECK_SYNC("hl_matrix_row_sum failed");
}

void hl_matrix_row_max(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  hl_matrix_row_op(aggregate::max(),
                   A_d,
                   C_d,
                   dimM,
                   dimN);
  CHECK_SYNC("hl_matrix_row_max failed");
}

void hl_matrix_row_min(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  hl_matrix_row_op(aggregate::min(),
                   A_d,
                   C_d,
                   dimM,
                   dimN);
  CHECK_SYNC("hl_matrix_row_min failed");
}

/**
 * @brief   matrix column operator.
 */
template<class Agg>
__global__ void KeMatrixColumnOp(Agg agg,
                                 real *E,
                                 real *Sum,
                                 int dimM,
                                 int dimN) {
  int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
  real tmp = agg.init();
  if (rowIdx < dimN) {
    for (int index = 0; index < dimM; index++) {
      tmp = agg(tmp, E[dimN * index + rowIdx]);
    }
    Sum[rowIdx] = tmp;
  }
}

template<class Agg, int blockDimX, int blockDimY>
__global__ void KeMatrixColumnOp_S(Agg agg,
                                   real *E,
                                   real *Sum,
                                   int dimM,
                                   int dimN) {
    __shared__ real _sum[blockDimX*blockDimY];
    int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = threadIdx.y;

  real tmp = agg.init();
  if (rowIdx < dimN) {
    for (; index < dimM;) {
      tmp = agg(tmp, E[dimN * index + rowIdx]);
      index += blockDimY;
    }
  }
  _sum[threadIdx.x + threadIdx.y*blockDimX] = tmp;
  __syncthreads();

  if (rowIdx < dimN) {
    if (threadIdx.y ==0) {
      real tmp = agg.init();
      for (int i=0; i < blockDimY; i++) {
        tmp = agg(tmp, _sum[threadIdx.x + i*blockDimX]);
      }
      Sum[rowIdx] = tmp;
    }
  }
}

template <class Agg>
void hl_matrix_column_op(Agg agg,
                         real *A_d,
                         real *C_d,
                         int dimM,
                         int dimN) {
  if (dimN >= 8192) {
    int blocksX = (dimN + 128 -1) / 128;
    int blocksY = 1;
    dim3 threads(128, 1);
    dim3 grid(blocksX, blocksY);
    KeMatrixColumnOp<Agg><<< grid, threads, 0, STREAM_DEFAULT >>>
             (agg, A_d, C_d, dimM, dimN);
  } else {
    int blocksX = (dimN + 32 -1) / 32;
    int blocksY = 1;
    dim3 threads(32, 32);
    dim3 grid(blocksX, blocksY);
    KeMatrixColumnOp_S<Agg, 32, 32><<< grid, threads, 0, STREAM_DEFAULT>>>
             (agg, A_d, C_d, dimM, dimN);
  }

  return;
}

void hl_matrix_column_sum(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  hl_matrix_column_op(aggregate::sum(),
                      A_d,
                      C_d,
                      dimM,
                      dimN);

  CHECK_SYNC("hl_matrix_column_sum failed");
}

void hl_matrix_column_max(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  hl_matrix_column_op(aggregate::max(),
                      A_d,
                      C_d,
                      dimM,
                      dimN);

  CHECK_SYNC("hl_matrix_column_max failed");
}

void hl_matrix_column_min(real *A_d, real *C_d, int dimM, int dimN) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_d);

  hl_matrix_column_op(aggregate::min(),
                      A_d,
                      C_d,
                      dimM,
                      dimN);

  CHECK_SYNC("hl_matrix_column_min failed");
}

template <int blockSize>
__global__ void KeVectorSum(real *E, real *Sum, int dimM) {
  __shared__ double sum_s[blockSize];
  int tid = threadIdx.x;
  int index = blockIdx.y*blockDim.x+threadIdx.x;

  sum_s[tid] = 0.0f;
  while (index < dimM) {
    sum_s[tid] += E[index];
    index += blockDim.x*gridDim.y;
  }
  __syncthreads();

  for (int stride = blockSize/2; stride > 0; stride = stride/2) {
    if (tid < stride) {
      sum_s[tid] += sum_s[tid + stride];
    }
    __syncthreads();
  }
  __syncthreads();

  if (tid == 0) {
    Sum[blockIdx.y] = sum_s[0];
  }
}

void hl_vector_sum(real *A_d, real *C_h, int dimM) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_h);

  int blockSize = 128;
  int gridSize = 128;
  int blocksX = 1;
  int blocksY = gridSize;
  dim3 threads(blockSize, 1);
  dim3 grid(blocksX, blocksY);

  struct _hl_event_st hl_event_st  = {.cu_event = t_resource.event};
  hl_event_t hl_event = &hl_event_st;
  while (!hl_cuda_event_is_ready(hl_event)) {}

  KeVectorSum<128><<< grid, threads, 0, STREAM_DEFAULT >>>
           (A_d, t_resource.gpu_mem, dimM);
  KeVectorSum<128><<< 1, threads, 0, STREAM_DEFAULT >>>
           (t_resource.gpu_mem, t_resource.cpu_mem, 128);

  hl_memcpy_async(C_h, t_resource.cpu_mem, sizeof(real), HPPL_STREAM_DEFAULT);
  hl_stream_record_event(HPPL_STREAM_DEFAULT, hl_event);

  hl_stream_synchronize(HPPL_STREAM_DEFAULT);
  hipError_t err = (hipError_t)hl_get_device_last_error();
  CHECK_EQ(hipSuccess, err)
    << "CUDA error: " << hl_get_device_error_string((size_t)err);
}

template <int blockSize>
__global__ void KeVectorAbsSum(real *E, real *Sum, int dimM) {
  __shared__ double sum_s[blockSize];
  int tid = threadIdx.x;
  int index = blockIdx.y*blockDim.x+threadIdx.x;

  sum_s[tid] = 0.0f;
  while (index < dimM) {
    sum_s[tid] += abs(E[index]);
    index += blockDim.x*gridDim.y;
  }
  __syncthreads();

  for (int stride = blockSize/2; stride > 0; stride = stride/2) {
    if (tid < stride) {
      sum_s[tid] += sum_s[tid + stride];
    }
    __syncthreads();
  }
  __syncthreads();

  if (tid == 0) {
    Sum[blockIdx.y] = sum_s[0];
  }
}

void hl_vector_abs_sum(real *A_d, real *C_h, int dimM) {
  CHECK_NOTNULL(A_d);
  CHECK_NOTNULL(C_h);

  int blockSize = 128;
  int gridSize = 128;
  int blocksX = 1;
  int blocksY = gridSize;
  dim3 threads(blockSize, 1);
  dim3 grid(blocksX, blocksY);

  struct _hl_event_st hl_event_st  = {.cu_event = t_resource.event};
  hl_event_t hl_event = &hl_event_st;
  while (!hl_cuda_event_is_ready(hl_event)) {}

  KeVectorAbsSum<128><<< grid, threads, 0, STREAM_DEFAULT >>>
           (A_d, t_resource.gpu_mem, dimM);
  KeVectorAbsSum<128><<< 1, threads, 0, STREAM_DEFAULT >>>
           (t_resource.gpu_mem, t_resource.cpu_mem, 128);

  hl_memcpy_async(C_h, t_resource.cpu_mem, sizeof(real), HPPL_STREAM_DEFAULT);
  hl_stream_record_event(HPPL_STREAM_DEFAULT, hl_event);

  hl_stream_synchronize(HPPL_STREAM_DEFAULT);
  hipError_t err = (hipError_t)hl_get_device_last_error();
  CHECK_EQ(hipSuccess, err)
    << "CUDA error: " << hl_get_device_error_string((size_t)err);
}
