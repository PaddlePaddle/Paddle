#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "hl_device_functions.cuh"
#include "paddle/utils/Logging.h"

__global__ void KeMaxSequenceForward(real *input,
                                     const int *sequence,
                                     real* output,
                                     int *index,
                                     int numSequences,
                                     int dim) {
  int dimIdx = threadIdx.x;
  int sequenceId = blockIdx.x;
  if (sequenceId >= numSequences) return;
  int start = sequence[sequenceId];
  int end = sequence[sequenceId+1];

  for (int i = dimIdx; i < dim; i += blockDim.x) {
    real tmp = -HL_FLOAT_MAX;
    int tmpId = -1;
    for (int insId = start; insId < end; insId++) {
      if (tmp < input[insId*dim + i]) {
        tmp = input[insId*dim + i];
        tmpId = insId;
      }
    }
    output[sequenceId*dim + i] = tmp;
    index[sequenceId*dim + i] = tmpId;
  }
}

void hl_max_sequence_forward(real* input,
                             const int* sequence,
                             real* output,
                             int *index,
                             int numSequences,
                             int dim) {
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(output);
  CHECK_NOTNULL(index);

  dim3 threads(256, 1);
  dim3 grid(numSequences, 1);
  KeMaxSequenceForward<<< grid, threads, 0, STREAM_DEFAULT >>>
      (input, sequence, output, index, numSequences, dim);
  CHECK_SYNC("hl_max_sequence_forward failed");
}

__global__ void KeMaxSequenceBackward(real *outputGrad,
                                      int *index,
                                      real* inputGrad,
                                      int numSequences,
                                      int dim) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int colIdx = idx % dim;
  if (idx < numSequences*dim) {
    int insId = index[idx];
    inputGrad[insId * dim + colIdx] += outputGrad[idx];
  }
}

void hl_max_sequence_backward(real* outputGrad,
                              int *index,
                              real* inputGrad,
                              int numSequences,
                              int dim) {
  CHECK_NOTNULL(outputGrad);
  CHECK_NOTNULL(index);
  CHECK_NOTNULL(inputGrad);

  unsigned int blocks = (numSequences * dim + 128 - 1) / 128;
  dim3 threads(128, 1);
  dim3 grid(blocks, 1);
  KeMaxSequenceBackward<<< grid, threads, 0, STREAM_DEFAULT >>>
      (outputGrad, index, inputGrad, numSequences, dim);
  CHECK_SYNC("hl_max_sequence_backward failed");
}

template<int blockDimX, int blockDimY, int gridDimX, bool AddRow>
__global__ void KeMatrixAddRows(real* output,
                                real* table,
                                int* ids,
                                int numSamples,
                                int tableSize,
                                int dim) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int sampleId = blockIdx.x + idy * gridDimX;

  while (sampleId < numSamples) {
    int tableId = ids[sampleId];
    if ((0 <= tableId) && (tableId < tableSize)) {
      real *outputData = output + sampleId * dim;
      real *tableData = table + tableId * dim;
      for (int i = idx; i < dim; i += blockDimX) {
        if (AddRow == 0) {
          outputData[i] += tableData[i];
        } else {
          paddle::paddleAtomicAdd(&tableData[i], outputData[i]);
        }
      }
    }
    sampleId += blockDimY*gridDimX;
  }
}

template<int blockDimX, int blockDimY, int gridDimX, bool seq2batch, bool isAdd>
__global__
void KeSequence2Batch(real *batch,
                      real *sequence,
                      const int *batchIndex,
                      int seqWidth,
                      int batchCount) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id = blockIdx.x + idy * gridDimX;
  while (id < batchCount) {
    int seqId = batchIndex[id];
    real* batchData = batch + id*seqWidth;
    real* seqData = sequence + seqId*seqWidth;
    for (int i = idx; i < seqWidth; i += blockDimX) {
      if (seq2batch) {
        if (isAdd) {
          batchData[i] += seqData[i];
        } else {
          batchData[i] = seqData[i];
        }
      } else {
        if (isAdd) {
          seqData[i] += batchData[i];
        } else {
          seqData[i] = batchData[i];
        }
      }
    }
    id += blockDimY*gridDimX;
  }
}

void hl_sequence2batch_copy(real *batch,
                            real *sequence,
                            const int *batchIndex,
                            int seqWidth,
                            int batchCount,
                            bool seq2batch) {
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(batch);
  CHECK_NOTNULL(batchIndex);

  dim3 threads(128, 8);
  dim3 grid(8, 1);
  if (seq2batch) {
    KeSequence2Batch<128, 8, 8, 1, 0><<< grid, threads, 0, STREAM_DEFAULT >>>
      (batch, sequence, batchIndex, seqWidth, batchCount);
  } else {
    KeSequence2Batch<128, 8, 8, 0, 0><<< grid, threads, 0, STREAM_DEFAULT >>>
      (batch, sequence, batchIndex, seqWidth, batchCount);
  }
  CHECK_SYNC("hl_sequence2batch_copy failed");
}

void hl_sequence2batch_add(real *batch,
                           real *sequence,
                           int *batchIndex,
                           int seqWidth,
                           int batchCount,
                           bool seq2batch) {
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(batch);
  CHECK_NOTNULL(batchIndex);

  dim3 threads(128, 8);
  dim3 grid(8, 1);
  if (seq2batch) {
    KeSequence2Batch<128, 8, 8, 1, 1><<< grid, threads, 0, STREAM_DEFAULT >>>
      (batch, sequence, batchIndex, seqWidth, batchCount);
  } else {
    KeSequence2Batch<128, 8, 8, 0, 1><<< grid, threads, 0, STREAM_DEFAULT >>>
      (batch, sequence, batchIndex, seqWidth, batchCount);
  }
  CHECK_SYNC("hl_sequence2batch_add failed");
}

template<bool normByTimes, bool seq2batch>
__global__
void KeSequence2BatchPadding(real* batch,
                             real* sequence,
                             const int* sequenceStartPositions,
                             const size_t sequenceWidth,
                             const size_t maxSequenceLength,
                             const size_t numSequences) {
  int batchIdx = blockIdx.y;
  int sequenceStart = sequenceStartPositions[batchIdx];
  int sequenceLength = sequenceStartPositions[batchIdx + 1] - sequenceStart;

  int sequenceIdx = blockIdx.x * blockDim.y + threadIdx.y;
  int batchBaseIdx = (sequenceIdx * numSequences + batchIdx) * sequenceWidth;
  int sequenceBaseIdx = (sequenceStart + sequenceIdx) * sequenceWidth;

  real scale = normByTimes ? (1.0f / (real)sequenceLength) : 1.0f;

  if (sequenceIdx < sequenceLength) {
    if (seq2batch) {
      /* sequence -> batch */
      for (int i = threadIdx.x; i < sequenceWidth; i += blockDim.x) {
        batch[batchBaseIdx + i] = scale * sequence[sequenceBaseIdx + i];
      }
    } else {
      /* batch -> sequence */
      for (int i = threadIdx.x; i < sequenceWidth; i += blockDim.x) {
        sequence[sequenceBaseIdx + i] = scale * batch[batchBaseIdx + i];
      }
    }
  } else if (sequenceIdx < maxSequenceLength) {
    if (seq2batch) {
      /* sequence -> batch */
      for (int i = threadIdx.x; i < sequenceWidth; i += blockDim.x) {
        batch[batchBaseIdx + i] = 0;
      }
    }
  }
}

void hl_sequence2batch_copy_padding(real* batch,
                                    real* sequence,
                                    const int* sequenceStartPositions,
                                    const size_t sequenceWidth,
                                    const size_t maxSequenceLength,
                                    const size_t numSequences,
                                    bool normByTimes,
                                    bool seq2batch) {
  CHECK_NOTNULL(batch);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(sequenceStartPositions);

  if (!normByTimes && numSequences == 1) {
    size_t elementCount = maxSequenceLength * sequenceWidth;
    if (seq2batch) {
      /* sequence -> batch */
      hl_memcpy_device2device(batch, sequence, sizeof(real) * elementCount);
    } else {
      /* batch -> sequence */
      hl_memcpy_device2device(sequence, batch, sizeof(real) * elementCount);
    }
    return;
  }

  const int CUDA_BLOCK_SIZE = 512;

  /* At least use 32 threads to copy sequenceWidth elements,
     and at least 8 elements for each thread. */
  int blockDimX = ((((sequenceWidth + 7) >> 3) + 31) >> 5) << 5;
  blockDimX = (blockDimX < CUDA_BLOCK_SIZE) ? blockDimX : CUDA_BLOCK_SIZE;

  int blockDimY = CUDA_BLOCK_SIZE / blockDimX;
  dim3 threads(blockDimX, blockDimY);

  int gridDimX = (maxSequenceLength * blockDimX + CUDA_BLOCK_SIZE - 1) /
      CUDA_BLOCK_SIZE;
  int gridDimY = numSequences;
  dim3 grid(gridDimX, gridDimY);

  if (seq2batch) {
    /* sequence -> batch */
    if (normByTimes) {
      KeSequence2BatchPadding<1, 1><<< grid, threads, 0, STREAM_DEFAULT >>>(
              batch, sequence, sequenceStartPositions,
              sequenceWidth, maxSequenceLength, numSequences);
    } else {
      KeSequence2BatchPadding<0, 1><<< grid, threads, 0, STREAM_DEFAULT >>>(
              batch, sequence, sequenceStartPositions,
              sequenceWidth, maxSequenceLength, numSequences);
    }
  } else {
    /* batch -> sequence */
    if (normByTimes) {
      KeSequence2BatchPadding<1, 0><<< grid, threads, 0, STREAM_DEFAULT >>>(
              batch, sequence, sequenceStartPositions,
              sequenceWidth, maxSequenceLength, numSequences);
    } else {
      KeSequence2BatchPadding<0, 0><<< grid, threads, 0, STREAM_DEFAULT >>>(
              batch, sequence, sequenceStartPositions,
              sequenceWidth, maxSequenceLength, numSequences);
    }
  }

  CHECK_SYNC("hl_sequence2batch_copy_padding failed");
}

__device__ inline float my_rsqrt(float x) {
  return rsqrtf(x);
}

__device__ inline double my_rsqrt(double x) {
  return rsqrt(x);
}

__global__ void KeSequenceAvgForward(real* dst,
                                     real* src,
                                     const int* starts,
                                     int height,
                                     int width,
                                     const int mode) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int row = gid / width;
  int col = gid % width;

  if (gid < height * width) {
    int start = starts[row];
    int end = starts[row + 1];
    int seqLength = end - start;
    if (seqLength == 0) return;
    real sum = 0.0;
    for (int i = 0; i < seqLength; i++) {
      sum += src[(start + i) * width + col];
    }
    sum = mode == 1 ? sum :
        (mode == 0 ? sum / seqLength : sum * my_rsqrt((real)seqLength));
    dst[row * width + col] = sum;
  }
}

void hl_sequence_avg_forward(real* dst,
                             real* src,
                             const int* starts,
                             int height,
                             int width,
                             const int mode) {
  CHECK_NOTNULL(dst);
  CHECK_NOTNULL(src);
  CHECK_NOTNULL(starts);

  int block = 512;
  int grid = DIVUP(width * height, 512);

  CHECK(mode == 0 || mode == 1 || mode == 2)
    << "mode error in hl_sequence_avg_forward!";

  KeSequenceAvgForward<<< grid, block, 0, STREAM_DEFAULT >>>
           (dst, src, starts, height, width, mode);
  CHECK_SYNC("hl_sequence_avg_forward failed");
}
