#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */


#include <cmath>
#include <stdlib.h>
#include "hl_cuda.h"
#include "hl_time.h"
#include "hl_base.h"
#include "hl_perturbation_util.cuh"

#define _USE_MATH_DEFINES

/*
 * Get the original coordinate for a pixel in a transformed image.
 * x, y: coordiate in the transformed image.
 * tgtCenter: the center coordiate of the transformed image.
 * imgSCenter: the center coordinate of the source image.
 * centerX, centerY: translation.
 * sourceX, sourceY: output coordinates in the original image.
 */
__device__ void getTranformCoord(int x, int y, real theta, real scale,
                                 real tgtCenter, real imgCenter,
                                 real centerR, real centerC,
                                 int* sourceX, int* sourceY) {
  real H[4] = {cosf(-theta), -sinf(-theta), sinf(-theta), cosf(-theta)};

  // compute coornidates in the rotated and scaled image
  real x_new = x - tgtCenter + centerC;
  real y_new = y - tgtCenter + centerR;

  // compute coornidates in the original image
  x_new -= imgCenter;
  y_new -= imgCenter;
  real xx = H[0] * x_new + H[1] * y_new;
  real yy = H[2] * x_new + H[3] * y_new;
  *sourceX = __float2int_rn(xx / scale + imgCenter);
  *sourceY = __float2int_rn(yy / scale + imgCenter);
}

/*
 * imgs:            (numImages, imgPixels)
 * target:          (numImages * samplingRate, tgtPixels)
 * the channels of one pixel are stored continuously in memory.
 *
 * created by Wei Xu (genome), converted by Jiang Wang
 */

__global__ void kSamplingPatches(const real* imgs, real* targets,
                                 int imgSize, int tgtSize, const int channels,
                                 int samplingRate, const real* thetas,
                                 const real* scales, const int* centerRs,
                                 const int* centerCs, const real padValue,
                                 const int numImages) {
  const int caseIdx = blockIdx.x * 4 + threadIdx.x;
  const int pxIdx = blockIdx.y * 128 + threadIdx.y;
  const int imgPixels = imgSize * imgSize;
  const int tgtPixels = tgtSize * tgtSize;
  const int numPatches = numImages * samplingRate;

  real tgtCenter = (tgtSize - 1) / 2;
  real imgCenter = (imgSize - 1) / 2;

  if (pxIdx < tgtPixels && caseIdx < numPatches) {
    const int imgIdx = caseIdx / samplingRate;

    // transform coordiates
    const int pxX = pxIdx % tgtSize;
    const int pxY = pxIdx / tgtSize;

    int srcPxX, srcPxY;
    getTranformCoord(pxX, pxY, thetas[imgIdx], scales[imgIdx], tgtCenter,
                     imgCenter, centerCs[caseIdx], centerRs[caseIdx], &srcPxX,
                     &srcPxY);

    imgs += (imgIdx * imgPixels + srcPxY * imgSize + srcPxX) * channels;
    targets += (caseIdx * tgtPixels + pxIdx) * channels;
    if (srcPxX >= 0 && srcPxX < imgSize && srcPxY >= 0 && srcPxY < imgSize) {
      for (int j = 0; j < channels; j++) targets[j] = imgs[j];
    } else {
      for (int j = 0; j < channels; j++) targets[j] = padValue;
    }
  }
}

/*
 * Functionality: generate the disturb (rotation and scaling) and
 *                sampling location sequence
 *
 * created by Wei Xu
 */
void hl_generate_disturb_params(real*& gpuAngle, real*& gpuScaleRatio,
                                int*& gpuCenterR, int*& gpuCenterC,
                                int numImages, int imgSize, real rotateAngle,
                                real scaleRatio, int samplingRate,
                                bool isTrain) {
  // The number of output samples.
  int numPatches = numImages * samplingRate;

  // create CPU perturbation parameters.
  real* r_angle = new real[numImages];
  real* s_ratio = new real[numImages];
  int* center_r = new int[numPatches];
  int* center_c = new int[numPatches];

  // generate the random disturbance sequence and the sampling locations
  if (isTrain) {  // random sampling for training
    // generate rotation ans scaling parameters
    // TODO(yuyang18): Since it will initialize random seed here, we can use
    // rand_r instead of rand to make this method thread safe.
    srand(getCurrentTimeStick());
    for (int i = 0; i < numImages; i++) {
      r_angle[i] =
          (rotateAngle * M_PI / 180.0) * (rand() / (RAND_MAX + 1.0)  // NOLINT
                                          - 0.5);
      s_ratio[i] =
          1 + (rand() / (RAND_MAX + 1.0) - 0.5) * scaleRatio;  // NOLINT
    }

    int imgCenter = (imgSize - 1) / 2;

    // generate sampling location parameters
    for (int i = 0; i < numImages; i++) {
      int j = 0;
      srand((unsigned)time(NULL));
      while (j < samplingRate) {
        int pxX =
            (int)(real(imgSize - 1) * rand() / (RAND_MAX + 1.0));  // NOLINT
        int pxY =
            (int)(real(imgSize - 1) * rand() / (RAND_MAX + 1.0));  // NOLINT

        const real H[4] = {cos(-r_angle[i]), -sin(-r_angle[i]),
                           sin(-r_angle[i]), cos(-r_angle[i])};
        real x = pxX - imgCenter;
        real y = pxY - imgCenter;
        real xx = H[0] * x + H[1] * y;
        real yy = H[2] * x + H[3] * y;

        real srcPxX = xx / s_ratio[i] + imgCenter;
        real srcPxY = yy / s_ratio[i] + imgCenter;

        if (srcPxX >= 0 && srcPxX <= imgSize - 1 && srcPxY >= 0 &&
            srcPxY <= imgSize - 1) {
          center_r[i * samplingRate + j] = pxY;
          center_c[i * samplingRate + j] = pxX;
          j++;
        }
      }
    }
  } else {  // central crop for testing
    for (int i = 0; i < numImages; i++) {
      r_angle[i] = 0.0;
      s_ratio[i] = 1.0;

      for (int j = 0; j < samplingRate; j++) {
        center_r[i * samplingRate + j] = (imgSize - 1) / 2;
        center_c[i * samplingRate + j] = (imgSize - 1) / 2;
      }
    }
  }

  // copy disturbance sequence to gpu
  hl_memcpy_host2device(gpuAngle, r_angle, sizeof(real) * numImages);
  hl_memcpy_host2device(gpuScaleRatio, s_ratio, sizeof(real) * numImages);

  delete[] r_angle;
  delete[] s_ratio;

  // copy sampling location sequence to gpu
  hl_memcpy_host2device(gpuCenterR, center_r, sizeof(int) * numPatches);
  hl_memcpy_host2device(gpuCenterC, center_c, sizeof(int) * numPatches);

  delete[] center_r;
  delete[] center_c;
}

void hl_conv_random_disturb_with_params(const real* images, int imgSize,
                                        int tgtSize, int channels,
                                        int numImages, int samplingRate,
                                        const real* gpuRotationAngle,
                                        const real* gpuScaleRatio,
                                        const int* gpuCenterR,
                                        const int* gpuCenterC,
                                        int paddingValue,
                                        real* target) {
  // The number of output samples.
  int numPatches = numImages * samplingRate;
  // The memory size of one output patch.
  int targetSize = tgtSize * tgtSize;

  dim3 threadsPerBlock(4, 128);
  dim3 numBlocks(DIVUP(numPatches, 4), DIVUP(targetSize, 128));

  kSamplingPatches <<<numBlocks, threadsPerBlock>>>
      (images, target, imgSize, tgtSize, channels, samplingRate,
      gpuRotationAngle, gpuScaleRatio, gpuCenterR, gpuCenterC,
      paddingValue, numImages);

  hl_device_synchronize();
}

void hl_conv_random_disturb(const real* images, int imgSize,
                            int tgtSize, int channels, int numImages,
                            real scaleRatio, real rotateAngle,
                            int samplingRate, real* gpu_r_angle,
                            real* gpu_s_ratio, int* gpu_center_r,
                            int* gpu_center_c, int paddingValue,
                            bool isTrain, real* targets) {
  // generate the random disturbance sequence and the sampling locations
  hl_generate_disturb_params(gpu_r_angle, gpu_s_ratio, gpu_center_r,
                  gpu_center_c, numImages, imgSize, rotateAngle,
                  scaleRatio, samplingRate, isTrain);

  hl_conv_random_disturb_with_params(
                  images, imgSize, tgtSize, channels, numImages,
                  samplingRate, gpu_r_angle, gpu_s_ratio,
                  gpu_center_r, gpu_center_r, paddingValue,
                  targets);
}
