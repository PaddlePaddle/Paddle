#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */


#include <float.h>
#include "hl_base.h"
#include "hl_cnn.h"
#include "hl_device_functions.cuh"

__global__ void KeFeature2col(size_t n, size_t height, const real* data_im,
                              size_t blockH, size_t blockW, size_t width,
                              size_t strideH, size_t strideW,
                              size_t paddingH, size_t paddingW,
                              size_t height_col, size_t width_col,
                              real* data_col) {
  size_t index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < n) {
    size_t w_out = index % width_col;
    index /= width_col;
    size_t h_out = index % height_col;
    size_t channel_in = index / height_col;
    size_t channel_out = channel_in * blockH * blockW;
    size_t h_in = h_out * strideH;
    size_t w_in = w_out * strideW;

    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    for (size_t i = 0; i < blockH; ++i) {
      for (size_t j = 0; j < blockW; ++j) {
        int rIdx = int(h_in+i);
        int cIdx = int(w_in+j);
        if ((rIdx-(int)paddingH) >= (int)height ||
            (rIdx-(int)paddingH) < 0 ||
            (cIdx-(int)paddingW) >= (int)width ||
            (cIdx-(int)paddingW) < 0) {
          *data_col = 0;
        } else {
          rIdx = rIdx + channel_in*height - paddingH;
          cIdx = cIdx - paddingW;
          *data_col = data_im[rIdx* width + cIdx];
        }
        data_col += height_col * width_col;
      }
    }
  }
}

void hl_expand_feature2col(const real* dataIm, size_t channels,
                           size_t height, size_t width,
                           size_t blockH, size_t blockW,
                           size_t strideH, size_t strideW,
                           size_t paddingH, size_t paddingW,
                           size_t outputH, size_t outputW,
                           real* dataCol) {
  size_t numKernels = channels * outputH * outputW;

  size_t blocks = (numKernels + 1024 -1) / 1024;
  size_t blockX = 512;
  size_t blockY = (blocks+512-1)/512;
  dim3 threads(1024, 1);
  dim3 grid(blockX, blockY);
  KeFeature2col<<< grid, threads, 0, STREAM_DEFAULT >>>
           (numKernels, height, dataIm, blockH, blockW, width,
           strideH, strideW, paddingH, paddingW,
           outputH, outputW, dataCol);
  CHECK_SYNC("hl_expand_feature2col failed");
}

__global__ void KeCol2Feature(size_t n, const real* data_col, size_t height,
                              size_t width, size_t channels,
                              size_t blockH, size_t blockW,
                              size_t strideH, size_t strideW,
                              size_t paddingH, size_t paddingW,
                              size_t height_col, size_t width_col,
                              real* data_im, real alpha, real beta) {
  size_t index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < n) {
    real val = 0;
    int w = int(index % width);
    int h = int((index / width) % height);
    int c = int(index / (width * height));
    if ((w - (int)paddingW) >= 0 &&
        (w - (int)paddingW) < (width-2 * paddingW) &&
        (h - (int)paddingH) >= 0 &&
        (h - paddingH) < (height - 2 * paddingH)) {
      // compute the start and end of the output
      int w_col_start =
        (w < (int)blockW) ? 0 : (w - int(blockW)) / (int)strideW + 1;
      int w_col_end =
        min((int)(w / (int)strideW + 1), (int)(width_col));
      int h_col_start =
        (h < (int)blockH) ? 0 : (h - (int)blockH) / (int)strideH + 1;
      int h_col_end = min(int(h / strideH + 1), int(height_col));
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          // the col location: [c * width * height + h_out, w_out]
          int c_col = int(c * blockH* blockW) + \
            (h - h_col * (int)strideH) * (int)blockW +
            (w - w_col * (int)strideW);
          val += data_col[(c_col * height_col + h_col) * width_col + w_col];
        }
      }
      h -= paddingH;
      w -= paddingW;
      real tD = data_im[c*((width-2*paddingW) * (height-2*paddingH)) +
                          h*(width-2*paddingW) + w];
      data_im[c*((width-2*paddingW) * (height-2*paddingH)) +
              h*(width-2*paddingW) + w] = alpha * val + beta*tD;
    }
  }
}

void hl_shrink_col2feature(const real * dataCol, size_t channels,
                           size_t height, size_t width,
                           size_t blockH, size_t blockW,
                           size_t strideH, size_t strideW,
                           size_t paddingH, size_t paddingW,
                           size_t outputH, size_t outputW,
                           real* dataIm, real alpha, real beta) {
  size_t numKernels = channels * (height + 2*paddingH) * (width + 2*paddingW);

  size_t blocks = (numKernels + 1024 -1) / 1024;
  size_t blockX = 512;
  size_t blockY = (blocks+512-1)/512;
  dim3 threads(1024, 1);
  dim3 grid(blockX, blockY);

  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  KeCol2Feature<<< grid, threads, 0, STREAM_DEFAULT >>>
           (numKernels, dataCol, height + 2*paddingH, width + 2*paddingW,
           channels, blockH, blockW, strideH, strideW, paddingH, paddingW,
           outputH, outputW, dataIm, alpha, beta);
  CHECK_SYNC("hl_shrink_col2feature failed");
}

__global__ void KeMaxPoolForward(const int nthreads, const real* inputData,
                                 const int channels, const int height,
                                 const int width,
                                 const int pooledH, const int pooledW,
                                 const int ksizeW, const int ksizeH,
                                 const int strideH, const int strideW,
                                 const int offsetH, const int offsetW,
                                 real* tgtData, const int tgtStride) {
  int index =  blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int pw = index % pooledW;
    int ph = (index / pooledW) % pooledH;
    int c = (index / pooledW / pooledH) % channels;
    int frameNum = index / pooledW / pooledH / channels;
    int hstart = ph * strideH - offsetH;
    int wstart = pw * strideW - offsetW;
    int hend = min(hstart + ksizeH, height);
    int wend = min(wstart + ksizeW, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    real maxval = -FLT_MAX;
    inputData += (frameNum * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (maxval < inputData[h * width + w])
          maxval = inputData[h * width + w];
      }
    }
    int tgtIndex = index % (pooledW * pooledH * channels) +
        frameNum * tgtStride;
    tgtData[tgtIndex] = maxval;
  }
}

void hl_maxpool_forward(const int frameCnt, const real* inputData,
                        const int channels,
                        const int height, const int width,
                        const int pooledH, const int pooledW,
                        const int sizeX, const int sizeY,
                        const int strideH, const int strideW,
                        const int paddingH, const int paddingW,
                        real* tgtData, const int tgtStride) {

  int num_kernels = pooledH * pooledW * channels * frameCnt;
  int blocks = (num_kernels + 1024 - 1) / 1024;
  dim3 threads(1024, 1);
  dim3 grid(blocks, 1);

  KeMaxPoolForward<<< grid, threads, 0, STREAM_DEFAULT >>>
           (num_kernels, inputData, channels, height, width,
           pooledH, pooledW, sizeX, sizeY, strideH, strideW,
           paddingH, paddingW, tgtData, tgtStride);
  CHECK_SYNC("hl_maxpool_forward failed");
}

__global__ void KeMaxPoolBackward(const int nthreads, const real* inputData,
                                  const real* outData, const real* outGrad,
                                  const int channels, const int height,
                                  const int width,
                                  const int pooledH, const int pooledW,
                                  const int sizeX, const int sizeY,
                                  const int strideH, const int strideW,
                                  const int padH, const int padW,
                                  real scaleA, real scaleB,
                                  real* targetGrad, const int outStride) {
  int index = blockIdx.x  * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int offsetW = index % width + padW;
    int offsetH = (index / width) % height + padH;
    int offsetC = (index / width / height) % channels;

    int frameNum = index / width / height / channels;
    int phstart = (offsetH < sizeY) ? 0 : (offsetH - sizeY) / strideH + 1;
    int pwstart = (offsetW < sizeX) ? 0 : (offsetW - sizeX) / strideW + 1;
    int phend = offsetH >= 0 ? min(offsetH / strideH + 1, pooledH) : 0;
    int pwend = offsetW >= 0 ? min(offsetW / strideW + 1, pooledW) : 0;
    real gradient = 0;
    real input = inputData[index];
    outData += (frameNum * outStride + offsetC * pooledH * pooledW);
    outGrad += (frameNum * outStride + offsetC * pooledH * pooledW);
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (input == outData[ph * pooledW + pw]) {
          gradient += outGrad[ph * pooledW + pw];
        }
      }
    }
    targetGrad[index] =
      scaleB * targetGrad[index] + scaleA * gradient;
  }
}

void hl_maxpool_backward(const int frameCnt, const real* inputData,
                        const real* outData, const real* outGrad,
                        const int channels, const int height,
                        const int width,
                        const int pooledH, const int pooledW,
                        const int sizeX, const int sizeY,
                        const int strideH, const int strideW,
                        const int paddingH, const int paddingW,
                        real scaleA, real scaleB,
                        real* targetGrad, const int outStride) {

  int num_kernels = height * width * channels * frameCnt;
  int blocks = (num_kernels + 1024 - 1) / 1024;

  KeMaxPoolBackward<<< blocks, 1024, 0, STREAM_DEFAULT >>>
           (num_kernels, inputData, outData, outGrad, channels,
           height, width, pooledH, pooledW, sizeX, sizeY,
           strideH, strideW,
           paddingH, paddingW,
           scaleA, scaleB,
           targetGrad, outStride);
  CHECK_SYNC("hl_maxpool_backward");
}

__global__ void KeAvgPoolForward(const int nthreads, const real* inputData,
                                 const int channels,
                                 const int height, const int width,
                                 const int pooledH, const int pooledW,
                                 const int sizeX, const int sizeY,
                                 const int strideH, const int strideW,
                                 const int padH, const int padW,
                                 real* tgtData, const int tgtStride) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int pw = index % pooledW;
    int ph = (index / pooledW) % pooledH;
    int c = (index / pooledW / pooledH) % channels;
    int frameNum = index / pooledW / pooledH / channels;

    int hstart = ph * strideH - padH;
    int wstart = pw * strideW - padW;
    int hend = min(hstart + sizeY, height + padH);
    int wend = min(wstart + sizeX, width + padW);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);

    real aveval = 0;
    inputData += (frameNum * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += inputData[h * width + w];
      }
    }
    int tgtIndex = index % (pooledW * pooledH * channels) +
        frameNum * tgtStride;
    tgtData[tgtIndex] = aveval / pool_size;
  }
}

void hl_avgpool_forward(const int frameCnt, const real* inputData,
                        const int channels,
                        const int height, const int width,
                        const int pooledH, const int pooledW,
                        const int sizeX, const int sizeY,
                        const int strideH, const int strideW,
                        const int paddingH, const int paddingW, 
                        real* tgtData, const int tgtStride) {
  int num_kernels = pooledH * pooledW * channels * frameCnt;
  int blocks = (num_kernels + 1024 - 1) / 1024;
  KeAvgPoolForward<<< blocks, 1024, 0, STREAM_DEFAULT >>>
           (num_kernels, inputData, channels,
           height, width, pooledH, pooledW,
           sizeX, sizeY, strideH, strideW,
           paddingH, paddingW, tgtData, tgtStride);
  CHECK_SYNC("hl_avgpool_forward failed");
}

__global__ void KeAvgPoolBackward(const int nthreads, const real* outGrad,
                                  const int channels, const int height,
                                  const int width,
                                  const int pooledH, const int pooledW,
                                  const int sizeX, const int sizeY,
                                  const int strideH, const int strideW,
                                  const int padH, const int padW,
                                  real scaleA, real scaleB,
                                  real* tgtGrad, const int outStride) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int offsetW = index % width + padW;
    int offsetH = (index / width) % height + padH;
    int offsetC = (index / width / height) % channels;
    int frameNum = index / width / height / channels;

    int phstart = (offsetH < sizeY) ? 0 : (offsetH - sizeY) / strideH + 1;
    int pwstart = (offsetW < sizeX) ? 0 : (offsetW - sizeX) / strideW + 1;
    int phend = offsetH >= 0 ? min(offsetH / strideH + 1, pooledH) : 0;
    int pwend = offsetW >= 0 ? min(offsetW / strideW + 1, pooledW) : 0;
    real gradient = 0;
    outGrad += (frameNum * outStride + offsetC * pooledH * pooledW);


    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * strideH - padH;
        int wstart = pw * strideW - padW;
        int hend = min(hstart + sizeY, height + padH);
        int wend = min(wstart + sizeX, width + padW);
        int poolsize = (hend - hstart) * (wend - wstart);
        gradient += outGrad[ph * pooledW + pw]/poolsize;
      }
    }
    tgtGrad[index] = scaleB * tgtGrad[index] + scaleA * gradient;
  }
}

void hl_avgpool_backward(const int frameCnt, const real* outGrad,
                         const int channels,
                         const int height, const int width,
                         const int pooledH, const int pooledW,
                         const int sizeX, const int sizeY,
                         const int strideH, const int strideW,
                         const int paddingH, const int paddingW,
                         real scaleA, real scaleB,
                         real* backGrad, const int outStride) {
  int num_kernels = height * width * channels * frameCnt;
  int blocks = (num_kernels + 1024 - 1) / 1024;

  KeAvgPoolBackward <<< blocks, 1024, 0, STREAM_DEFAULT >>>
           (num_kernels, outGrad, channels, height, width,
           pooledH, pooledW, sizeX, sizeY,
           strideH, strideW,
           paddingH, paddingW,
           scaleA, scaleB,
           backGrad, outStride);
  CHECK_SYNC("hl_avgpool_backward failed");
}

__global__ void KeCMRNormFillScale(size_t nthreads, const real* in,
                                   real* scale, size_t channels,
                                   size_t height, size_t width, size_t size,
                                   real alpha) {
  size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local offset
    size_t w = index % width;
    size_t h = (index / width) % height;
    size_t n = index / width / height;
    size_t offset = (n * channels * height + h) * width + w;
    size_t step = height * width;
    in += offset;
    scale += offset;
    size_t head = 0;
    size_t pre_pad = (size - 1) / 2;
    size_t post_pad = size - pre_pad - 1;
    real accum_scale = 0;
    // fill the scale at [n, :, h, w]
    // accumulate values
    while (head < post_pad) {
      accum_scale += in[head * step] * in[head * step];
      ++head;
    }
    // until we reach size, nothing needs to be subtracted
    while (head < size) {
      accum_scale += in[head * step] * in[head * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha;
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_scale += in[head * step] * in[head * step];
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha;
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha;
      ++head;
    }
  }
}

 __global__ void KeCMRNormOutput(size_t nthreads, const real* in,
                                 const real* scale, real negative_beta,
                                 real* out) {
  size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    out[index] = in[index] * pow(scale[index], negative_beta);
  }
}

void hl_CMRNorm_forward(size_t frameCnt, const real* in, real* scale,
                        real* out, size_t channels,
                        size_t height, size_t width, size_t sizeX,
                        real alpha, real beta) {
  size_t threadsNum = frameCnt * height * width;
  size_t blocksX = (threadsNum + 1024 - 1) / 1024;
  size_t blocksY = 1;
  dim3 threads(1024, 1);
  dim3 grid(blocksX, blocksY);

  KeCMRNormFillScale<<<grid, threads, 0, STREAM_DEFAULT>>>
      (threadsNum, in, scale, channels, height, width, sizeX, alpha);

  threadsNum = frameCnt * height * width *channels;
  blocksX = (threadsNum + 1024 -1) / 1024;
  dim3 threads2(1024, 1);
  dim3 grid2(blocksX, blocksY);
  KeCMRNormOutput<<<grid2, threads2, 0, STREAM_DEFAULT>>>
           (threadsNum, in, scale, beta, out);
  CHECK_SYNC("hl_CMRNorm_forward");
}

__global__ void KeCMRNormDiff(size_t nthreads, const real* bottom_data,
                              const real* top_data, const real* scale,
                              const real* top_diff, size_t channels,
                              size_t height, size_t width, size_t size,
                              real negative_beta, real cache_ratio,
                              real* bottom_diff ) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local offset
    size_t w = index % width;
    size_t h = (index / width) % height;
    size_t n = index / width / height;
    size_t offset = (n * channels * height + h) * width + w;
    size_t step = height * width;
    bottom_data += offset;
    top_data += offset;
    scale += offset;
    top_diff += offset;
    bottom_diff += offset;
    int head = 0;
    int pre_pad = size - (size + 1) / 2;
    int post_pad = size - pre_pad - 1;
    real accum_ratio = 0;
    // accumulate values
    while (head < post_pad) {
      accum_ratio += top_diff[head * step] *
        top_data[head * step] / scale[head * step];
      ++head;
    }
    // until we reach size, nothing needs to be subtracted
    while (head < size) {
      accum_ratio += top_diff[head * step] *
        top_data[head * step] / scale[head * step];
      bottom_diff[(head - post_pad) * step] +=
        top_diff[(head - post_pad) * step] *
        pow(scale[(head - post_pad) * step], negative_beta) - cache_ratio *
        bottom_data[(head - post_pad) * step] * accum_ratio;
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_ratio += top_diff[head * step] * top_data[head * step] /
          scale[head * step];
      accum_ratio -= top_diff[(head - size) * step] *
          top_data[(head - size) * step] / scale[(head - size) * step];
      bottom_diff[(head - post_pad) * step] +=
        top_diff[(head - post_pad) * step] *
        pow(scale[(head - post_pad) * step], negative_beta) - cache_ratio *
        bottom_data[(head - post_pad) * step] * accum_ratio;
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      accum_ratio -= top_diff[(head - size) * step] *
          top_data[(head - size) * step] / scale[(head - size) * step];
      bottom_diff[(head - post_pad) * step] +=
        top_diff[(head - post_pad) * step] *
        pow(scale[(head - post_pad) * step], negative_beta) - cache_ratio *
        bottom_data[(head - post_pad) * step] * accum_ratio;
      ++head;
    }
  }
}

void hl_CMRNorm_backward(size_t frameCnt, const real* inV,
                         const real* scale,
                         const real* outV, const real* outDiff,
                         real *inDiff, size_t channels,
                         size_t height, size_t width, size_t sizeX,
                         real alpha, real beta) {
  size_t threadsNum = frameCnt * height * width;
  size_t blocksX = (threadsNum + 1024 - 1) / 1024;
  size_t blocksY = 1;
  dim3 threads(1024, 1);
  dim3 grid(blocksX, blocksY);
  KeCMRNormDiff <<<grid, threads, 0, STREAM_DEFAULT>>>
           (threadsNum, inV, outV, scale, outDiff, channels,
           height, width, sizeX, alpha, beta, inDiff);
  CHECK_SYNC("hl_CMRNorm_backward");
}

__global__ void KeBilinearInterpFw(const real* in,
                                   const size_t inImgH,
                                   const size_t inImgW,
                                   const size_t inputH,
                                   const size_t inputW,
                                   real* out,
                                   const size_t outImgH,
                                   const size_t outImgW,
                                   const size_t outputH,
                                   const size_t outputW,
                                   const size_t numChannels,
                                   const real ratioH,
                                   const real ratioW) {
  int nthreads = outputH * outputW;                      
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < nthreads) {
    int outIdH = tid / outputW;
    int outIdW = tid % outputW;
    int inImgSize = inputW / numChannels;
    int outImgSize = outputW / numChannels;
    int channelId = outIdW / outImgSize;

    int outImgIdy = (outIdW % outImgSize) / outImgW;
    int inImgIdy = ratioH * outImgIdy;
    int hId = (inImgIdy < inImgH - 1) ? 1 : 0;
    real h1lambda = ratioH * outImgIdy - inImgIdy;
    real h2lambda = 1.f - h1lambda;

    int outImgIdx = tid % outImgW;
    int inImgIdx = ratioW * outImgIdx;
    int wId = (inImgIdx < inImgW - 1) ? 1 : 0;
    real w1lambda = ratioW * outImgIdx - inImgIdx;
    real w2lambda = 1.f - w1lambda;

    const real* inPos =
      &in[outIdH * inputW + channelId * inImgSize + inImgIdy * inImgW + inImgIdx];

    // bilinear interpolation
    out[outIdH * outputW + outIdW] =
      h2lambda * (w2lambda * inPos[0]            + w1lambda * inPos[wId]) + 
      h1lambda * (w2lambda * inPos[hId * inImgW] + w1lambda * inPos[hId * inImgW + wId]);
  }
}

void hl_bilinear_forward(const real* inData,
                         const size_t inImgH,
                         const size_t inImgW,
                         const size_t inputH,
                         const size_t inputW,
                         real* outData,
                         const size_t outImgH,
                         const size_t outImgW,
                         const size_t outputH,
                         const size_t outputW,
                         const size_t numChannels,
                         const real ratioH,
                         const real ratioW) {
  int threadNum = outputH * outputW;
  int blocks = (threadNum + 1024 - 1) / 1024;

  KeBilinearInterpFw<<< blocks, 1024, 0, STREAM_DEFAULT>>>(
    inData, inImgH, inImgW, inputH, inputW, outData, outImgH,
    outImgW, outputH, outputW, numChannels, ratioH, ratioW);
  CHECK_SYNC("hl_bilinear_forward failed");
}

__global__ void KeBilinearInterpBw(real* in,
                                   const size_t inImgH,
                                   const size_t inImgW,
                                   const size_t inputH,
                                   const size_t inputW,
                                   const real* out,
                                   const size_t outImgH,
                                   const size_t outImgW,
                                   const size_t outputH,
                                   const size_t outputW,
                                   const size_t numChannels,
                                   const real ratioH,
                                   const real ratioW) {
  int nthreads = outputH * outputW;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < nthreads) {
    int outIdH = tid / outputW;
    int outIdW = tid % outputW;
    int inImgSize = inputW / numChannels;
    int outImgSize = outputW / numChannels;
    int channelId = outIdW / outImgSize;

    int outImgIdy = (outIdW % outImgSize) / outImgW;
    int inImgIdy = ratioH * outImgIdy;
    int hId = (inImgIdy < inImgH - 1) ? 1 : 0;
    real h1lambda = ratioH * outImgIdy - inImgIdy;
    real h2lambda = 1.f - h1lambda;

    int outImgIdx = tid % outImgW;
    int inImgIdx = ratioW * outImgIdx;
    int wId = (inImgIdx < inImgW - 1) ? 1 : 0;
    real w1lambda = ratioW * outImgIdx - inImgIdx;
    real w2lambda = 1.f - w1lambda;

    real* inPos =
      &in[outIdH * inputW + channelId * inImgSize + inImgIdy * inImgW + inImgIdx];
    const real* outPos = &out[outIdH * outputW + outIdW];
    paddle::paddleAtomicAdd(&inPos[0], h2lambda * w2lambda * outPos[0]);
    paddle::paddleAtomicAdd(&inPos[wId], h2lambda * w1lambda * outPos[0]);
    paddle::paddleAtomicAdd(&inPos[hId * inImgW], h1lambda * w2lambda * outPos[0]);
    paddle::paddleAtomicAdd(&inPos[hId * inImgW + wId], h1lambda * w1lambda * outPos[0]);
  }
}

void hl_bilinear_backward(real* inGrad,
                          const size_t inImgH,
                          const size_t inImgW,
                          const size_t inputH,
                          const size_t inputW,
                          const real* outGrad,
                          const size_t outImgH,
                          const size_t outImgW,
                          const size_t outputH,
                          const size_t outputW,
                          const size_t numChannels,
                          const real ratioH,
                          const real ratioW) {
  int threadNum = outputH * outputW;
  int blocks = (threadNum + 1024 - 1) / 1024;

  KeBilinearInterpBw<<< blocks, 1024, 0, STREAM_DEFAULT>>>(
    inGrad, inImgH, inImgW, inputH, inputW, outGrad, outImgH,
    outImgW, outputH, outputW, numChannels, ratioH, ratioW);
  CHECK_SYNC("hl_bilinear_backward failed");
}

__global__ void maxoutFpCompute(size_t nthreads, const real * inData,
                                real * outData, int* idData, 
                                size_t size, size_t featLen, size_t groups) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < nthreads) {
    size_t batch_idx = index / size;
    size_t i = index % size;
    size_t channel_idx = i / featLen;
    size_t feat_idx = i % featLen;
    size_t data_idx = (batch_idx * size + channel_idx * featLen) * groups + feat_idx;
    real max = inData[data_idx];
    int maxId = 0;
    for (size_t g = 1; g < groups; ++g) {
      real tmp = inData[data_idx + g * featLen];
      if (tmp > max) {
        max = tmp;
        maxId = g;
      }
    }
    outData[index] = max;
    idData[index] = maxId;
  }
}

void hl_maxout_forward(const real* inData, real* outData,
                       int* idData, size_t batchSize, size_t size,
                       size_t featLen, size_t groups) {
  int num_kernels = size * batchSize;
  int blocks = (num_kernels + 1024 - 1) / 1024;
  maxoutFpCompute<<< blocks, 1024, 0, STREAM_DEFAULT>>>(
    num_kernels, inData, outData, idData, size, featLen, groups);
  CHECK_SYNC("hl_maxout_forward failed");
}

__global__ void maxoutBpCompute(size_t nthreads, real* inGrad,
                                const real* outGrad, const int* idData,
                                size_t size, size_t featLen, size_t groups) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < nthreads) {
    size_t batch_idx = index / size;
    size_t i = index % size;
    size_t channel_idx = i / featLen;
    size_t feat_idx = i % featLen;
    size_t newIndex = batch_idx * size;
    size_t gradIdx = (channel_idx * groups + (idData + newIndex)[i]) * featLen + feat_idx;
    (inGrad + newIndex * groups)[gradIdx] += (outGrad + newIndex)[i];
  }
}

void hl_maxout_backward(real* inGrad, const real* outGrad,
                        const int* idData, size_t batchSize, size_t size,
                        size_t featLen, size_t groups) {
  int num_kernels = size * batchSize;
  int blocks = (num_kernels + 1024 - 1) / 1024;
  maxoutBpCompute<<< blocks, 1024, 0, STREAM_DEFAULT >>>(
    num_kernels, inGrad, outGrad, idData, size, featLen, groups);
  CHECK_SYNC("hl_maxout_backward failed");
}
