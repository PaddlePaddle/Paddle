#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */


#include "hl_base.h"
#include "hl_cuda_cublas.h"
#include "hl_device_functions.cuh"
#include "hl_activation_functions.h"
#include "paddle/utils/Logging.h"

typedef hppl::Active<real>::forward  t_forward;
typedef hppl::Active<real>::backward t_backward;

bool hl_lstm_sequence_parallel(int frameSize) {
  if (frameSize == 32 || frameSize == 64) {
    return true;
  } else {
    return false;
  }
}

class frameValue {
public:
  real *value_;
  __device__ frameValue(real *value) : value_(value) {}
  template <int reversed, int frameSize>
  __device__ inline void init(int start, int length, int idx) {
    if (reversed == 0) {
      value_ += start * frameSize + idx;
    } else {
      value_ += (start + length - 1) * frameSize + idx;
    }
  }
  __device__ inline real *getPtr() const {return value_;}
  __device__ inline real getValue() {return *value_;}
  __device__ inline void setValue(real value) {*value_ = value;}
  template <int reversed, int frameSize>
  __device__ inline void nextFrame() {
    if (reversed == 0) {
      value_ += frameSize;
    } else {
      value_ -= frameSize;
    }
  }
};

__device__ __forceinline__
void ptx_sync(const int id, const int barriers) {
  asm volatile("bar.sync %0, %1;" : : "r"(id), "r"(barriers) : "memory");
}

__device__ __forceinline__
void ptx_arrive(const int id, const int barriers) {
  asm volatile("bar.arrive %0, %1;" : : "r"(id), "r"(barriers) : "memory");
}

template<int valueSize, int frameSize>
__device__ __forceinline__ real
forward_sequence(real value,
                 real *shValue,
                 real *state,
                 real *preOutput,
                 real *output,
                 real check,
                 int index,
                 t_forward activeNode,
                 t_forward activeGate,
                 t_forward activeState) {
  real out;
  real prevOut;
  real state_r;
  const int idx = index % frameSize;
  const int idy = index / frameSize;
  // assert(index < valueSize);

  if (idy == 0) {
    value = activeNode(value);
    shValue[index] = value;
  }
  if (idy == 1 || idy == 2) {
    state_r = state[idx];
    value += state_r * check;
    value = activeGate(value);
    shValue[index] = value;
  }
  ptx_sync(1, valueSize);
  if (idy == 3) {
    state_r = state[idx];
    state_r = state_r * shValue[idx + frameSize * 2];
    state_r += shValue[idx] * shValue[idx + frameSize];
    state[idx] = state_r;
    ptx_arrive(2, frameSize * 2);
    value += state_r * check;
    value = activeGate(value);
    shValue[index] = value;
    ptx_sync(3, frameSize * 2);
    prevOut = preOutput[idx];
    out = prevOut * value;
    output[idx] = out;
  }
  if (idy == 0) {
    ptx_sync(2, frameSize * 2);
    prevOut = state[idx];
     prevOut = activeState(prevOut);
    preOutput[idx] = prevOut;
    ptx_arrive(3, frameSize * 2);
  }
  return value;
}

#define     OUTPUT_BARRIER_ID               10
#define     OUTPUT_BARRIER_ID2              11
template<int valueSize, int frameSize, int reversed,
         int computeThreads, int blockSize>
__global__ void KeLstmForward(real *gateValue,
                              real *state,
                              real *output,
                              real *preOutput,
                              real *checkIg,
                              real *checkFg,
                              real *checkOg,
                              real *weight,
                              const int *starts,
                              hl_activation_mode_t active_node,
                              hl_activation_mode_t active_gate,
                              hl_activation_mode_t active_state) {
  __shared__ real shValue[valueSize];
  __shared__ real shState[frameSize];
  __shared__ real shPrevOutput[frameSize];
  __shared__ real shOutput[frameSize];

  const int index = threadIdx.x;
  int start = starts[blockIdx.x];
  int length = starts[blockIdx.x + 1] - start;

  /* init */
  real check;
  real value;
  frameValue frameGate(gateValue);
  frameValue frameState(state);
  frameValue frameOutput(output);
  frameValue framePreOutput(preOutput);
  if (index < valueSize) {
    const int idx = index % frameSize;
    const int idy = index / frameSize;
    frameGate.init<reversed, valueSize>(start, length, index);
    value = frameGate.getValue();
    if (idy == 0) {
      shState[idx] = 0.0;
    } else if (idy == 1) {
      check = checkIg[idx];
    } else if (idy == 2) {
      check = checkFg[idx];
    } else if (idy == 3) {
      check = checkOg[idx];
    }

    if (idy == 3) {
      frameState.init<reversed, frameSize>(start, length, idx);
      frameOutput.init<reversed, frameSize>(start, length, idx);
      framePreOutput.init<reversed, frameSize>(start, length, idx);
    }

    ptx_sync(1, valueSize);
  }

  for (int i = 0; i < length; ++i) {
    if (index < valueSize) {
      if (valueSize == 128) {
        if (i != 0) {
          ptx_sync(OUTPUT_BARRIER_ID2, blockSize);
          value += shValue[index];
        }
      }
      value = forward_sequence<valueSize, frameSize>(
        value, shValue, shState, shPrevOutput, shOutput, check, index,
        hppl::gpu::forward[active_node],
        hppl::gpu::forward[active_gate],
        hppl::gpu::forward[active_state]);
      const int idx = index % frameSize;
      const int idy = index / frameSize;
      if (valueSize == 128) {
        if (idy == 3) {
          ptx_arrive(OUTPUT_BARRIER_ID, frameSize + 128);
        }
      }
      if (valueSize == 256) {
        ptx_sync(OUTPUT_BARRIER_ID, valueSize);
      }
      frameGate.setValue(value);
      if (idy == 3) {
        frameState.setValue(shState[idx]);
        frameOutput.setValue(shOutput[idx]);
        framePreOutput.setValue(shPrevOutput[idx]);
        frameState.nextFrame<reversed, frameSize>();
        frameOutput.nextFrame<reversed, frameSize>();
        framePreOutput.nextFrame<reversed, frameSize>();
      }
      if (i != length - 1) {
        frameGate.nextFrame<reversed, valueSize>();
        value = frameGate.getValue();
      }
    }
    if (i != length - 1) {
      if (valueSize == 128) {
        if (valueSize <= index) {
          real B_r[frameSize];
          const int computeIdx = index - valueSize;
          if (i == 0) {
            #pragma unroll
            for (int n = 0; n < frameSize; n++) {
              B_r[n] = weight[n * valueSize + computeIdx];
            }
          }
          ptx_sync(OUTPUT_BARRIER_ID, frameSize + 128);
          real A_r[frameSize];
          for (int n = 0; n < frameSize; n++) {
            A_r[n] = shOutput[n];
          }
          real sum = 0.0f;
          for (int n = 0; n < frameSize; n++) {
            sum += A_r[n]*B_r[n];
          }
          shValue[computeIdx] = sum;
          ptx_arrive(OUTPUT_BARRIER_ID2, blockSize);
        }
      }
      if (valueSize == 256) {
        real B_r[frameSize];
        if (i == 0) {
          #pragma unroll
          for (int n = 0; n < frameSize; n++) {
            B_r[n] = weight[n * valueSize + index];
          }
        }
        real sum = 0.0f;
        for (int n = 0; n < frameSize; n++) {
          sum += shOutput[n]*B_r[n];
        }
        value += sum;
      }
    }
  }
}

void hl_lstm_parallel_forward(real *gateValue,
                              real *stateValue,
                              real *preOutputValue,
                              real *outputValue,
                              real *checkIg,
                              real *checkFg,
                              real *checkOg,
                              real *weight,
                              const int *sequence,
                              int frameSize,
                              int numSequences,
                              bool reversed,
                              hl_activation_mode_t active_node,
                              hl_activation_mode_t active_gate,
                              hl_activation_mode_t active_state) {
  CHECK(frameSize == 32 || frameSize == 64);
  dim3 grid(numSequences, 1);
  if (!reversed) {
    if (frameSize == 32) {
      KeLstmForward<128, 32, 0, 128, 256>
               <<<grid, 256, 0, STREAM_DEFAULT>>>
               (gateValue, stateValue, outputValue, preOutputValue,
               checkIg, checkFg, checkOg, weight, sequence,
               active_node, active_gate, active_state);
    } else if (frameSize == 64) {
      KeLstmForward<256, 64, 0, 256, 256>
               <<<grid, 256, 0, STREAM_DEFAULT>>>
               (gateValue, stateValue, outputValue, preOutputValue,
               checkIg, checkFg, checkOg, weight, sequence,
               active_node, active_gate, active_state);
    }
  } else {
    if (frameSize == 32) {
      KeLstmForward<128, 32, 1, 128, 256>
               <<<grid, 256, 0, STREAM_DEFAULT>>>
               (gateValue, stateValue, outputValue, preOutputValue,
               checkIg, checkFg, checkOg, weight, sequence,
               active_node, active_gate, active_state);
    } else if (frameSize == 64) {
      KeLstmForward<256, 64, 1, 256, 256>
               <<<grid, 256, 0, STREAM_DEFAULT>>>
               (gateValue, stateValue, outputValue, preOutputValue,
               checkIg, checkFg, checkOg, weight, sequence,
               active_node, active_gate, active_state);
    }
  }
  CHECK_SYNC("hl_lstm_parallel_forward failed");
}

__device__ __forceinline__
void transpose_32x32(real a[], const int idx) {
  int addr = idx % 32;
  #pragma unroll
  for (int k = 1; k < 32; k++) {
    // rSrc[k] = __shfl(rSrc[k], (threadIdx.x + k) % 32, 32);
    addr = __shfl(addr, (idx + 1) % 32, 32);
    a[k] = __shfl(a[k], addr, 32);
  }

  #pragma unroll
  for (int tid = 0; tid < 31; tid++) {
    real tmp = (idx > tid) ? a[0] : a[1];
    #pragma unroll
    for (int k = 31; k > 0; k--) {
      a[(k + 1) % 32] = (idx > tid) ? a[k] : a[(k + 1) % 32];
    }
    a[1] = tmp;
  }

  addr = (32 - idx) % 32;
  #pragma unroll
  for (int k = 0; k < 32; k++) {
    a[k] = __shfl(a[k], addr, 32);
    addr = __shfl(addr, (idx + 31) % 32, 32);
  }
}

template<int valueSize, int frameSize>
__device__ void
backward_sequence(real rGateValue,
                  real rOutputGrad,
                  real rPreOutputValue,
                  real &rGateGrad,
                  real &rStateGrad,
                  real *shStateGrad,
                  real *shStateValue,
                  real *shGateValue,
                  real rCheck,
                  real &rGateValuePrev,
                  int index,
                  t_backward activeNode,
                  t_backward activeGate,
                  t_backward activeState) {
  const int frameIdx = index % frameSize;
  const int frameIdy = index / frameSize;
  if (frameIdy == 3) {
    real rPrevOutputGrad;
    rPrevOutputGrad = rOutputGrad * rGateValue;
    rStateGrad = activeState(rPrevOutputGrad, rPreOutputValue);
    rGateGrad = rOutputGrad * rPreOutputValue;
    rGateGrad = activeGate(rGateGrad, rGateValue);
    rStateGrad += rGateGrad * rCheck;
    shStateGrad[index] = rStateGrad;
    ptx_arrive(3, valueSize);
  } else if (frameIdy == 1) {
    shGateValue[frameIdx + frameSize] = rGateValue;
    rStateGrad = rGateGrad * rCheck;
    shStateGrad[index] = rStateGrad;
    ptx_sync(3, valueSize);
    rStateGrad += shStateGrad[frameIdx + frameSize *2];
    rStateGrad += shStateGrad[frameIdx + frameSize *3];
    rGateGrad = rStateGrad * shGateValue[frameIdx];
    rGateGrad = activeGate(rGateGrad, rGateValue);
  } else if (frameIdy == 2) {
    rStateGrad = rStateGrad * rGateValuePrev;
    rStateGrad += rGateGrad * rCheck;
    shStateGrad[index] = rStateGrad;
    ptx_sync(3, valueSize);
    rStateGrad += shStateGrad[frameIdx + frameSize];
    rStateGrad += shStateGrad[frameIdx + frameSize *3];
    rGateValuePrev = rGateValue;
    rGateGrad = rStateGrad * shStateValue[frameIdx];
    rGateGrad = activeGate(rGateGrad, rGateValue);
  } else if (frameIdy == 0) {
    shGateValue[frameIdx] = rGateValue;
    ptx_sync(3, valueSize);
    rStateGrad = shStateGrad[frameIdx + frameSize];
    rStateGrad += shStateGrad[frameIdx + frameSize *2];
    rStateGrad += shStateGrad[frameIdx + frameSize *3];
    rGateGrad = rStateGrad * shGateValue[frameIdx + frameSize];
    rGateGrad = activeNode(rGateGrad, rGateValue);
  }
}

template<int valueSize, int frameSize>
__device__ void load_weight(real rWeight[], real *weight, const int index) {
  if (valueSize == 128) {
    weight += index;
    #pragma unroll
    for (int n = 0; n < frameSize; n++) {
      rWeight[n] = weight[n*valueSize];
    }
    transpose_32x32(rWeight, index % 32);
  }
  if (valueSize == 256) {
    int id = (index / 32) % 2;
    weight += index - id * 32 + id * 32 * valueSize;
    #pragma unroll
    for (int n = 0; n < 32; n++) {
      rWeight[n] = weight[n*valueSize];
      rWeight[n + 32] = weight[n*valueSize + 32];
    }
    transpose_32x32(rWeight, index % 32);
    transpose_32x32(&rWeight[32], index % 32);
  }
}

template<int valueSize, int frameSize, int reversed>
__global__ void KeLstmBackward(real *gateValue,
                               real *gateGrad,
                               real *stateValue,
                               real *stateGrad,       /* do not need save */
                               real *preOutputValue,
                               real *preOutputGrad,   /* do not need save */
                               real *checkIg,
                               real *checkIgGrad,
                               real *checkFg,
                               real *checkFgGrad,
                               real *checkOg,
                               real *checkOgGrad,
                               real *outputGrad,
                               real *weightValue,
                               const int *starts,
                               hl_activation_mode_t active_node,
                               hl_activation_mode_t active_gate,
                               hl_activation_mode_t active_state) {
  __shared__ real shGateValue[valueSize];
  __shared__ real shStateGrad[valueSize];
  __shared__ real shStateValue[frameSize];
  __shared__ real shGateGrad[4][frameSize];
  __shared__ real shOutputGrad[4][frameSize];
  const int index = threadIdx.x;
  int start = starts[blockIdx.x];
  int length = starts[blockIdx.x + 1] - start;

  const int frameIdx = index % frameSize;
  const int frameIdy = index / frameSize;
  real rCheck;
  real rCheckGrad;
  real rGateGrad;
  real rStateGrad;
  real rGateValuePrev;
  real rPreOutputValue;
  real rOutputGrad;
  real rGateValue;
  real rStateValue;

  frameValue frameGateValue(gateValue);
  frameValue frameGateGrad(gateGrad);
  frameValue framePreOutputValue(preOutputValue);
  frameValue frameStateValue(stateValue);
  frameValue frameOutputGrad(outputGrad);
  if (frameIdy == 0) {
  } else if (frameIdy == 1) {
    rCheck = checkIg[frameIdx];
  } else if (frameIdy == 2) {
    rCheck = checkFg[frameIdx];
    rGateValuePrev = 0.0;
    rStateGrad = 0.0;
  } else if (frameIdy == 3) {
    rCheck = checkOg[frameIdx];
    framePreOutputValue.init<!reversed, frameSize>(start, length, frameIdx);
    frameOutputGrad.init<!reversed, frameSize>(start, length, frameIdx);
    rOutputGrad = frameOutputGrad.getValue();
    rPreOutputValue = framePreOutputValue.getValue();
    frameStateValue.init<!reversed, frameSize>(start, length, frameIdx);
    rStateValue = frameStateValue.getValue();
  }

  frameGateValue.init<!reversed, valueSize>(start, length, index);
  frameGateGrad.init<!reversed, valueSize>(start, length, index);
  rGateValue = frameGateValue.getValue();
  rGateGrad = 0.0;
  rCheckGrad = 0.0;

  real B_r[frameSize];
  load_weight<valueSize, frameSize>(B_r, weightValue, index);

  for (int i = 0; i < length; ++i) {
    if (frameIdy == 3) {
      if (i != length -1) {
        frameStateValue.nextFrame<!reversed, frameSize>();
        shStateValue[frameIdx] = frameStateValue.getValue();
      } else {
        shStateValue[frameIdx] = 0.0;
      }
    }
    backward_sequence<valueSize, frameSize>(
        rGateValue, rOutputGrad, rPreOutputValue, rGateGrad,
        rStateGrad, shStateGrad, shStateValue, shGateValue,
        rCheck, rGateValuePrev, index,
        hppl::gpu::backward[active_node],
        hppl::gpu::backward[active_gate],
        hppl::gpu::backward[active_state]);
    if (frameIdy == 3) {
      rCheckGrad += rGateGrad * rStateValue;
      rStateValue = shStateValue[frameIdx];
    }

    frameGateGrad.setValue(rGateGrad);
    frameGateGrad.nextFrame<!reversed, valueSize>();

    if (i != length - 1) {
      if (frameIdy == 3) {
        framePreOutputValue.nextFrame<!reversed, frameSize>();
        rPreOutputValue = framePreOutputValue.getValue();
        frameOutputGrad.nextFrame<!reversed, frameSize>();
        rOutputGrad = frameOutputGrad.getValue();
      } else if (frameIdy == 2) {
        rCheckGrad += rGateGrad * shStateValue[frameIdx];
      } else if (frameIdy == 1) {
        rCheckGrad += rGateGrad * shStateValue[frameIdx];
      }

      frameGateValue.nextFrame<!reversed, valueSize>();
      rGateValue = frameGateValue.getValue();
      shGateGrad[frameIdy][frameIdx] = rGateGrad;
      if (valueSize == 128) {
        real sum = 0.0f;
        #pragma unroll
        for (int n = 0; n < frameSize; n++) {
          sum += shGateGrad[frameIdy][n]*B_r[n];
        }
        if (frameIdy == 3) {
          rOutputGrad += sum;
        } else {
          shOutputGrad[frameIdy][frameIdx] = sum;
        }
      }
      if (valueSize == 256) {
        ptx_sync(5, valueSize);
        real A_r[frameSize];
        for (int n = 0; n < frameSize; n++) {
          A_r[n] = shGateGrad[frameIdy][n];
        }
        real sum = 0.0f;
        for (int n = 0; n < frameSize; n++) {
          sum += A_r[n]*B_r[n];
        }
        if (frameIdy == 3) {
          rOutputGrad += sum;
        } else {
          shOutputGrad[frameIdy][frameIdx] = sum;
        }
      }

      if (frameIdy == 3) {
        ptx_sync(6, valueSize);
        #pragma unroll
        for (int i = 0; i < 3; i ++) {
          rOutputGrad += shOutputGrad[i][frameIdx];
        }
      } else {
        ptx_arrive(6, valueSize);
      }
    }
  }

  /* TODO: Temporary save & merger in another kernel */
  if (frameIdy == 1) {
    if (checkIgGrad) paddle::paddleAtomicAdd(checkIgGrad+frameIdx, rCheckGrad);
  } else if (frameIdy == 2) {
    if (checkFgGrad) paddle::paddleAtomicAdd(checkFgGrad+frameIdx, rCheckGrad);
  } else if (frameIdy == 3) {
    if (checkOgGrad) paddle::paddleAtomicAdd(checkOgGrad+frameIdx, rCheckGrad);
  }
}

void hl_lstm_parallel_backward_data(real *gateValue,
                                    real *gateGrad,
                                    real *stateValue,
                                    real *stateGrad,
                                    real *preOutputValue,
                                    real *preOutputGrad,
                                    real *outputGrad,
                                    real *checkIg,
                                    real *checkIgGrad,
                                    real *checkFg,
                                    real *checkFgGrad,
                                    real *checkOg,
                                    real *checkOgGrad,
                                    real *weight,
                                    const int *sequence,
                                    int frameSize,
                                    int numSequences,
                                    bool reversed,
                                    hl_activation_mode_t active_node,
                                    hl_activation_mode_t active_gate,
                                    hl_activation_mode_t active_state) {
  CHECK(frameSize == 32 || frameSize == 64 ||
        frameSize == 128 || frameSize == 256);
  dim3 grid(numSequences, 1);
  if (!reversed) {
    if (frameSize == 32) {
      KeLstmBackward<128, 32, 0><<<grid, 128, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    } else if (frameSize == 64) {
      KeLstmBackward<256, 64, 0><<<grid, 256, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    } else if (frameSize == 128) {
      KeLstmBackward<512, 128, 0><<<grid, 512, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    } else if (frameSize == 256) {
      KeLstmBackward<1024, 256, 0><<<grid, 1024, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    }
  } else {
    if (frameSize == 32) {
      KeLstmBackward<128, 32, 1><<<grid, 128, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    } else if (frameSize == 64) {
      KeLstmBackward<256, 64, 1><<<grid, 256, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    } else if (frameSize == 128) {
      KeLstmBackward<512, 128, 1><<<grid, 512, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    } else if (frameSize == 256) {
      KeLstmBackward<1024, 256, 1><<<grid, 1024, 0, STREAM_DEFAULT>>>
          (gateValue, gateGrad, stateValue, stateGrad, preOutputValue,
          preOutputGrad, checkIg, checkIgGrad, checkFg, checkFgGrad, checkOg,
          checkOgGrad, outputGrad, weight, sequence,
          active_node, active_gate, active_state);
    }
  }
  CHECK_SYNC("hl_lstm_parallel_backward_data");
}

template<int B_X, int B_Y>
__global__ void KeSetGradZero(real *gateGrad,
    const int *starts, int valueSize, int numSequences, bool reversed) {
  // const int tid = threadIdx.x;

  const int frameIdx = blockIdx.x * B_X + threadIdx.x;
  const int numSeqId = blockIdx.y * B_Y + threadIdx.y;

  if (numSeqId >= numSequences || frameIdx >= valueSize) return;

  if (!reversed) {
    int seqId = starts[numSeqId];
    gateGrad[seqId * valueSize + frameIdx] = 0.0;
  } else {
    int seqId = starts[numSeqId + 1] - 1;
    gateGrad[seqId * valueSize + frameIdx] = 0.0;
  }
}

void hl_lstm_parallel_backward_weight(real *weightGrad,
                                      real *outputValue,
                                      real *gateGrad,
                                      const int *sequence,
                                      int frameSize,
                                      int batchSize,
                                      int numSequences,
                                      bool reversed) {
  int valueSize = 4 * frameSize;
  dim3 threads(32, 32);
  dim3 grid((valueSize + 32 - 1) / 32, (numSequences + 32 - 1) / 32);
  KeSetGradZero<32, 32><<<grid, threads, 0, STREAM_DEFAULT>>>
           (gateGrad, sequence, valueSize, numSequences, reversed);

  if (!reversed) {
    hl_matrix_mul(outputValue,
      HPPL_OP_T, gateGrad + valueSize, HPPL_OP_N, weightGrad,
      frameSize, valueSize, batchSize - 1,
      1.0, 1.0);
  } else {
    hl_matrix_mul(outputValue + frameSize,
      HPPL_OP_T, gateGrad, HPPL_OP_N, weightGrad,
      frameSize, valueSize, batchSize - 1,
      1.0, 1.0);
  }
  CHECK_SYNC("hl_lstm_parallel_backward_weight");
}
