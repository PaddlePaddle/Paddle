#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "Im2Col.h"
#include "hl_device_functions.cuh"

namespace paddle {

template <class T>
__global__ void im2col(const T* data_im,
                       int numOuts,
                       int height,
                       int width,
                       int blockH,
                       int blockW,
                       int strideH,
                       int strideW,
                       int paddingH,
                       int paddingW,
                       int dilationH,
                       int dilationW,
                       int height_col,
                       int width_col,
                       T* data_col) {
  int index = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < numOuts) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * blockH * blockW;
    int h_in = h_out * strideH;
    int w_in = w_out * strideW;

    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    for (int i = 0; i < blockH; ++i) {
      for (int j = 0; j < blockW; ++j) {
        int rIdx = int(h_in + i * dilationH);
        int cIdx = int(w_in + j * dilationW);
        if ((rIdx - (int)paddingH) >= (int)height ||
            (rIdx - (int)paddingH) < 0 ||
            (cIdx - (int)paddingW) >= (int)width ||
            (cIdx - (int)paddingW) < 0) {
          *data_col = 0;
        } else {
          rIdx = rIdx + channel_in * height - paddingH;
          cIdx = cIdx - paddingW;
          *data_col = data_im[rIdx * width + cIdx];
        }
        data_col += height_col * width_col;
      }
    }
  }
}

/*
 * imShape = [inputChannels, inputHeight, inputWidth]
 * colShape =
 *   [inputChannels, filterHeight, filterWidth, outputHeight, outputWidth]
 */
template <class T>
class Im2ColFunctor<kCFO, DEVICE_TYPE_GPU, T> {
 public:
  void operator()(const T* imData,
                  const TensorShape& imShape,
                  T* colData,
                  const TensorShape& colShape,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth,
                  int dilationHeight,
                  int dilationWidth) {
    int inputChannels = imShape[0];
    int inputHeight = imShape[1];
    int inputWidth = imShape[2];
    int filterHeight = colShape[1];
    int filterWidth = colShape[2];
    int outputHeight = colShape[3];
    int outputWidth = colShape[4];

    int numKernels = inputChannels * outputHeight * outputWidth;
    int blocks = (numKernels + 1024 - 1) / 1024;
    int blockX = 512;
    int blockY = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);
    im2col<T><<<grid, threads, 0, STREAM_DEFAULT>>>(imData,
                                                    numKernels,
                                                    inputHeight,
                                                    inputWidth,
                                                    filterHeight,
                                                    filterWidth,
                                                    strideHeight,
                                                    strideWidth,
                                                    paddingHeight,
                                                    paddingWidth,
                                                    dilationHeight,
                                                    dilationWidth,
                                                    outputHeight,
                                                    outputWidth,
                                                    colData);
    CHECK_SYNC("Im2ColFunctor GPU failed");
  }
};

template <class T>
__global__ void col2im(size_t n,
                       const T* data_col,
                       size_t height,
                       size_t width,
                       size_t channels,
                       size_t blockH,
                       size_t blockW,
                       size_t strideH,
                       size_t strideW,
                       size_t paddingH,
                       size_t paddingW,
                       size_t dilationH,
                       size_t dilationW,
                       size_t height_col,
                       size_t width_col,
                       T* data_im) {
  size_t index =
      (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < n) {
    T val = 0;
    int w = int(index % width);
    int h = int((index / width) % height);
    int c = int(index / (width * height));
    int filterH = (blockH - 1) * dilationH + 1;
    int filterW = (blockW - 1) * dilationW + 1;

    if ((w - (int)paddingW) >= 0 &&
        (w - (int)paddingW) < (width - 2 * paddingW) &&
        (h - (int)paddingH) >= 0 && (h - paddingH) < (height - 2 * paddingH)) {
      // compute the start and end of the output
      int w_col_start =
          (w < (int)filterW) ? 0 : (w - int(filterW)) / (int)strideW + 1;
      int w_col_end = min((int)(w / (int)strideW + 1), (int)(width_col));
      int h_col_start =
          (h < (int)filterH) ? 0 : (h - (int)filterH) / (int)strideH + 1;
      int h_col_end = min(int(h / strideH + 1), int(height_col));

      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          // the col location: [c * width * height + h_out, w_out]
          int h_k = (h - h_col * strideH);
          int w_k = (w - w_col * strideW);
          if (h_k % dilationH == 0 && w_k % dilationW == 0) {
            h_k /= dilationH;
            w_k /= dilationW;
            int c_col =
                (((c * blockH + h_k) * blockW + w_k) * height_col + h_col) *
                    width_col +
                w_col;
            val += data_col[c_col];
          }
        }
      }
      h -= paddingH;
      w -= paddingW;
      data_im[c * ((width - 2 * paddingW) * (height - 2 * paddingH)) +
              h * (width - 2 * paddingW) + w] += val;
    }
  }
}

/*
 * imShape = [inputChannels, inputHeight, inputWidth]
 * colShape =
 *   [inputChannels, filterHeight, filterWidth, outputHeight, outputWidth]
 */
template <class T>
class Col2ImFunctor<kCFO, DEVICE_TYPE_GPU, T> {
 public:
  void operator()(T* imData,
                  const TensorShape& imShape,
                  const T* colData,
                  const TensorShape& colShape,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth,
                  int dilationHeight,
                  int dilationWidth) {
    int inputChannels = imShape[0];
    int inputHeight = imShape[1];
    int inputWidth = imShape[2];
    int filterHeight = colShape[1];
    int filterWidth = colShape[2];
    int outputHeight = colShape[3];
    int outputWidth = colShape[4];

    size_t numKernels = inputChannels * (inputHeight + 2 * paddingHeight) *
                        (inputWidth + 2 * paddingWidth);

    size_t blocks = (numKernels + 1024 - 1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);

    // To avoid involving atomic operations, we will launch one kernel per
    // bottom dimension, and then in the kernel add up the top dimensions.
    col2im<T><<<grid, threads, 0, STREAM_DEFAULT>>>(
        numKernels,
        colData,
        inputHeight + 2 * paddingHeight,
        inputWidth + 2 * paddingWidth,
        inputChannels,
        filterHeight,
        filterWidth,
        strideHeight,
        strideWidth,
        paddingHeight,
        paddingWidth,
        dilationHeight,
        dilationWidth,
        outputHeight,
        outputWidth,
        imData);
    CHECK_SYNC("Col2ImFunctor GPU failed");
  }
};

template class Im2ColFunctor<kCFO, DEVICE_TYPE_GPU, float>;
template class Im2ColFunctor<kCFO, DEVICE_TYPE_GPU, double>;
template class Col2ImFunctor<kCFO, DEVICE_TYPE_GPU, float>;
template class Col2ImFunctor<kCFO, DEVICE_TYPE_GPU, double>;

template <class T>
__global__ void im2colOCF(const T* imData,
                          T* colData,
                          int inputChannels,
                          int inputHeight,
                          int inputWidth,
                          int filterHeight,
                          int filterWidth,
                          int strideHeight,
                          int strideWidth,
                          int paddingHeight,
                          int paddingWidth,
                          int dilationHeight,
                          int dilationWidth,
                          int outputHeight,
                          int outputWidth) {
  int swId = blockIdx.x;
  int shId = blockIdx.y;
  for (int channelId = threadIdx.z; channelId < inputChannels;
       channelId += blockDim.z) {
    for (int idy = threadIdx.y; idy < filterHeight; idy += blockDim.y) {
      for (int idx = threadIdx.x; idx < filterWidth; idx += blockDim.x) {
        int widthOffset =
            idx * dilationHeight + swId * strideWidth - paddingWidth;
        int heightOffset =
            idy * dilationWidth + shId * strideHeight - paddingHeight;
        int imOffset = widthOffset + heightOffset * inputWidth +
                       channelId * inputHeight * inputWidth;

        int colOffset = idx + idy * filterWidth +
                        channelId * filterHeight * filterWidth +
                        (shId * outputWidth + swId) *
                            (inputChannels * filterHeight * filterWidth);

        if (heightOffset >= inputHeight || heightOffset < 0 ||
            widthOffset >= inputWidth || widthOffset < 0) {
          colData[colOffset] = T(0);
        } else {
          colData[colOffset] = imData[imOffset];
        }
      }
    }
  }
}

/*
 * imShape = [inputChannels, inputHeight, inputWidth]
 * colShape =
 *   [outputHeight, outputWidth, inputChannels, filterHeight, filterWidth]
 */
template <class T>
class Im2ColFunctor<kOCF, DEVICE_TYPE_GPU, T> {
 public:
  void operator()(const T* imData,
                  const TensorShape& imShape,
                  T* colData,
                  const TensorShape& colShape,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth,
                  int dilationHeight,
                  int dilationWidth) {
    int inputChannels = imShape[0];
    int inputHeight = imShape[1];
    int inputWidth = imShape[2];
    int filterHeight = colShape[3];
    int filterWidth = colShape[4];
    int outputHeight = colShape[0];
    int outputWidth = colShape[1];

    int blockDimX = 0;
    int blockDimY = 0;
    if (filterHeight <= 4 && filterWidth <= 4) {
      blockDimX = 4;
      blockDimY = 4;
    } else if (filterHeight <= 8 && filterWidth <= 8) {
      blockDimX = 8;
      blockDimY = 8;
    } else if (filterHeight <= 16 && filterWidth <= 16) {
      blockDimX = 16;
      blockDimY = 16;
    } else {
      blockDimX = 32;
      blockDimY = 32;
    }

    int blockDimZ = 1024 / blockDimX / blockDimY;
    dim3 threads(blockDimX, blockDimY, std::min(blockDimZ, inputChannels));
    dim3 grid(outputWidth, outputHeight);
    im2colOCF<T><<<grid, threads, 0, STREAM_DEFAULT>>>(imData,
                                                       colData,
                                                       inputChannels,
                                                       inputHeight,
                                                       inputWidth,
                                                       filterHeight,
                                                       filterWidth,
                                                       strideHeight,
                                                       strideWidth,
                                                       paddingHeight,
                                                       paddingWidth,
                                                       dilationHeight,
                                                       dilationWidth,
                                                       outputHeight,
                                                       outputWidth);
    CHECK_SYNC("Im2ColFunctor GPU failed");
  }
};

template <class T>
__global__ void col2imOCF(T* imData,
                          const T* colData,
                          int inputChannels,
                          int inputHeight,
                          int inputWidth,
                          int filterHeight,
                          int filterWidth,
                          int strideHeight,
                          int strideWidth,
                          int paddingHeight,
                          int paddingWidth,
                          int dilationHeight,
                          int dilationWidth,
                          int outputHeight,
                          int outputWidth) {
  int swId = blockIdx.x;
  int shId = blockIdx.y;
  for (int channelId = threadIdx.z; channelId < inputChannels;
       channelId += blockDim.z) {
    for (int idy = threadIdx.y; idy < filterHeight; idy += blockDim.y) {
      for (int idx = threadIdx.x; idx < filterWidth; idx += blockDim.x) {
        int widthOffset =
            idx * dilationWidth + swId * strideWidth - paddingWidth;
        int heightOffset =
            idy * dilationHeight + shId * strideHeight - paddingHeight;
        int imOffset = widthOffset + heightOffset * inputWidth +
                       channelId * inputHeight * inputWidth;

        int colOffset = idx + idy * filterWidth +
                        channelId * filterHeight * filterWidth +
                        (shId * outputWidth + swId) *
                            (inputChannels * filterHeight * filterWidth);

        if (heightOffset >= 0 && heightOffset < inputHeight &&
            widthOffset >= 0 && widthOffset < inputWidth) {
          paddle::paddleAtomicAdd(imData + imOffset, colData[colOffset]);
        }
      }
    }
  }
}

/*
 * imShape = [inputChannels, inputHeight, inputWidth]
 * colShape =
 *   [outputHeight, outputWidth, inputChannels, filterHeight, filterWidth]
 */
template <class T>
class Col2ImFunctor<kOCF, DEVICE_TYPE_GPU, T> {
 public:
  void operator()(T* imData,
                  const TensorShape& imShape,
                  const T* colData,
                  const TensorShape& colShape,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth,
                  int dilationHeight,
                  int dilationWidth) {
    int inputChannels = imShape[0];
    int inputHeight = imShape[1];
    int inputWidth = imShape[2];
    int filterHeight = colShape[3];
    int filterWidth = colShape[4];
    int outputHeight = colShape[0];
    int outputWidth = colShape[1];

    int blockDimX = 0;
    int blockDimY = 0;
    if (filterHeight <= 4 && filterWidth <= 4) {
      blockDimX = 4;
      blockDimY = 4;
    } else if (filterHeight <= 8 && filterWidth <= 8) {
      blockDimX = 8;
      blockDimY = 8;
    } else if (filterHeight <= 16 && filterWidth <= 16) {
      blockDimX = 16;
      blockDimY = 16;
    } else {
      blockDimX = 32;
      blockDimY = 32;
    }

    int blockDimZ = 1024 / blockDimX / blockDimY;
    dim3 threads(blockDimX, blockDimY, std::min(blockDimZ, inputChannels));
    dim3 grid(outputWidth, outputHeight);
    col2imOCF<T><<<grid, threads, 0, STREAM_DEFAULT>>>(imData,
                                                       colData,
                                                       inputChannels,
                                                       inputHeight,
                                                       inputWidth,
                                                       filterHeight,
                                                       filterWidth,
                                                       strideHeight,
                                                       strideWidth,
                                                       paddingHeight,
                                                       paddingWidth,
                                                       dilationHeight,
                                                       dilationWidth,
                                                       outputHeight,
                                                       outputWidth);
    CHECK_SYNC("Col2ImFunctor GPU failed");
  }
};

template class Im2ColFunctor<kOCF, DEVICE_TYPE_GPU, float>;
template class Im2ColFunctor<kOCF, DEVICE_TYPE_GPU, double>;
template class Col2ImFunctor<kOCF, DEVICE_TYPE_GPU, float>;
template class Col2ImFunctor<kOCF, DEVICE_TYPE_GPU, double>;

}  // namespace paddle
