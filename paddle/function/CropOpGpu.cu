#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "CropOp.h"
#include "hl_base.h"

namespace paddle {

__global__ void KeCrop(real* outputs,
                       const real* inputs,
                       int inC,
                       int inH,
                       int inW,
                       int cropC,
                       int cropH,
                       int cropW,
                       int outC,
                       int outH,
                       int outW,
                       int nthreads) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nthreads) {
    const int w = idx % outW;
    const int h = (idx / outW) % outH;
    const int c = (idx / outW / outH) % outC;
    const int n = idx / outW / outH / outC;

    const int off = ((n * inC + c + cropC) * inH + h + cropH) * inW + cropW + w;
    outputs[idx] = inputs[off];
  }
}

template <>
void Crop<DEVICE_TYPE_GPU>(real* outputs,
                           const real* inputs,
                           const TensorShape inShape,
                           const TensorShape outShape,
                           const FuncConfig& conf) {
  std::vector<uint32_t> crop_corner =
      conf.get<std::vector<uint32_t>>("crop_corner");
  int cropC = crop_corner[1];
  int cropH = crop_corner[2];
  int cropW = crop_corner[3];

  int num = inShape[0];
  int inC = inShape[1];
  int inH = inShape[2];
  int inW = inShape[3];

  int outC = outShape[1];
  int outH = outShape[2];
  int outW = outShape[3];

  size_t nth = num * outC * outH * outW;
  int blockSize = 1024;
  int gridSize = (nth + blockSize - 1) / blockSize;

  KeCrop<<<gridSize, blockSize, 0, STREAM_DEFAULT>>>(outputs,
                                                     inputs,
                                                     inC,
                                                     inH,
                                                     inW,
                                                     cropC,
                                                     cropH,
                                                     cropW,
                                                     outC,
                                                     outH,
                                                     outW,
                                                     nth);
  CHECK_SYNC("Crop");
}

__global__ void KeCropDiff(const real* inGrad,
                           real* outGrad,
                           int inC,
                           int inH,
                           int inW,
                           int cropC,
                           int cropH,
                           int cropW,
                           int outC,
                           int outH,
                           int outW,
                           int nthreads) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nthreads) {
    const int w = idx % inW;
    const int h = (idx / inW) % inH;
    const int c = (idx / inW / inH) % inC;
    const int n = idx / inW / inH / inC;

    const int off =
        ((n * outC + c + cropC) * outH + h + cropH) * outW + cropW + w;

    outGrad[off] += inGrad[idx];
  }
}

template <>
void CropGrad<DEVICE_TYPE_GPU>(const real* inGrad,
                               real* outGrad,
                               const TensorShape inShape,
                               const TensorShape outShape,
                               const FuncConfig& conf) {
  std::vector<uint32_t> crop_corner =
      conf.get<std::vector<uint32_t>>("crop_corner");
  int cropC = crop_corner[1];
  int cropH = crop_corner[2];
  int cropW = crop_corner[3];

  int num = outShape[0];
  int outC = outShape[1];
  int outH = outShape[2];
  int outW = outShape[3];

  int inC = inShape[1];
  int inH = inShape[2];
  int inW = inShape[3];

  size_t nth = num * inC * inH * inW;
  int blockSize = 1024;
  int gridSize = (nth + blockSize - 1) / blockSize;

  KeCropDiff<<<gridSize, blockSize, 0, STREAM_DEFAULT>>>(inGrad,
                                                         outGrad,
                                                         inC,
                                                         inH,
                                                         inW,
                                                         cropC,
                                                         cropH,
                                                         cropW,
                                                         outC,
                                                         outH,
                                                         outW,
                                                         nth);
  CHECK_SYNC("CropGrad");
}

}  // namespace paddle
