#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "ContextProjectionOp.h"

namespace paddle {

template <bool padding>
__global__ void KeContextProjectionForward(const real* input,
                                           const int* sequence,
                                           const real* weight,
                                           real* output,
                                           int input_dim,
                                           int context_length,
                                           int context_start,
                                           int begin_pad) {
  int idx = threadIdx.x;
  int block_size = blockDim.x;
  int sequenceId = blockIdx.x;
  int seq_start = sequence[sequenceId];
  int seq_end = sequence[sequenceId+1];
  real value = 0;

  int instances = seq_end - seq_start + context_length - 1;
  output += seq_start * input_dim * context_length;
  input += seq_start * input_dim;
  for (int k = 0; k <= input_dim / block_size; k++) {
    if (idx < input_dim) {
      for (int i = 0; i < instances; i++) {
        // i + context_start;
        if ((i + context_start) < 0) {
          if (padding) {
            value = weight[i * input_dim + idx];
          } else {
            continue;
          }
        } else if ((i + context_start) >= (seq_end - seq_start)) {
          if (padding) {
            value =
              weight[(begin_pad + i + context_start - (seq_end - seq_start)) *
                         input_dim + idx];
          } else {
            continue;
          }
        } else {
          value = input[(i + context_start) * input_dim + idx];
        }

        int outx = (i - context_length) < 0 ? i : (context_length - 1);
        int outy = (i - context_length) < 0 ? 0 : (i - (context_length - 1));
        real* output_r =
          output + outy * input_dim * context_length + outx * input_dim;
        for (int j = outy; j < seq_end - seq_start; j++) {
          output_r[idx] += value;
          if (j - outy == outx) break;
          output_r += (context_length - 1) * input_dim;
        }
      }
    }
    idx += block_size;
  }
}

/**
 * @brief   Context projection forward.
 *
 * @param[in]   input           input sequence.
 * @param[in]   sequence        sequence index.
 * @param[in]   weight          padding data.
 * @param[out]  output          output sequence.
 * @param[in]   num_sequences    number of sequences.
 * @param[in]   input_dim        input sequence dimension.
 * @param[in]   context_length   context length.
 * @param[in]   context_start    context start.
 * @param[in]   begin_pad        number of extra timesteps added at the
 * beginning.
 *
 */
void hl_context_projection_forward(const real* input,
                                   const int* sequence,
                                   const real* weight,
                                   real* output,
                                   size_t num_sequences,
                                   size_t input_dim,
                                   size_t context_length,
                                   int context_start,
                                   size_t begin_pad) {
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(output);

  int block_size = 128;
  int blocks_x = num_sequences;
  int blocks_y = 1;
  dim3 threads(block_size, 1);
  dim3 grid(blocks_x, blocks_y);

  if (weight) {
    KeContextProjectionForward<true><<< grid, threads, 0, STREAM_DEFAULT >>>
      (input, sequence, weight, output, input_dim,
       context_length, context_start, begin_pad);
  } else  {
    KeContextProjectionForward<false><<< grid, threads, 0, STREAM_DEFAULT >>>
      (input, sequence, weight, output, input_dim,
       context_length, context_start, begin_pad);
  }
  CHECK_SYNC("hl_context_projection_forward failed");
}

template <>
void ContextProjectionForward<DEVICE_TYPE_GPU>(GpuMatrix& output,
                                               const GpuMatrix& input,
                                               const GpuMatrix& weight,
                                               const GpuIVector& sequence,
                                               size_t context_length,
                                               int context_start,
                                               size_t begin_pad) {
  hl_context_projection_forward(input.getData(),
                                sequence.getData(),
                                weight ? weight.getData() : nullptr,
                                output.getData(),
                                sequence.getSize() - 1,
                                input.getWidth(),
                                context_length,
                                context_start,
                                begin_pad);
}

__global__ void KeContextProjectionBackwardData(const real* out_grad,
                                                const int* sequence,
                                                real* in_grad,
                                                size_t input_dim,
                                                int context_length,
                                                int context_start) {
  int idx = threadIdx.x;
  int block_size = blockDim.x;
  int sequenceId = blockIdx.x;
  int seq_start = sequence[sequenceId];
  int seq_end = sequence[sequenceId+1];
  real value = 0;

  int instances = seq_end - seq_start + context_length - 1;
  auto out = const_cast<real*>(out_grad);
  out += seq_start * input_dim * context_length;
  in_grad += seq_start * input_dim;
  for (int k = 0; k <= input_dim / block_size; k++) {
    if (idx < input_dim) {
      for (int i = 0; i < instances; i++) {
        if ((i + context_start) < 0) {
          continue;
        } else if ((i + context_start) >= (seq_end - seq_start)) {
          continue;
        } else {
          // value = 0;
          value = in_grad[(i + context_start) * input_dim + idx];
        }

        int outx = (i - context_length) < 0 ? i : (context_length - 1);
        int outy = (i - context_length) < 0 ? 0 : (i - (context_length - 1));
        real* output_r =
          out + outy * input_dim * context_length + outx * input_dim;
        for (int j = outy; j < seq_end - seq_start; j++) {
          value += output_r[idx];
          if (j - outy == outx) break;
          output_r += (context_length - 1) * input_dim;
        }
        in_grad[(i + context_start) * input_dim + idx] = value;
      }
    }
    idx += block_size;
  }
}

/**
 * @brief   Context projection backward data.
 *
 * @param[in]   out_grad         output gradient.
 * @param[in]   sequence         sequence index.
 * @param[out]  input_grad       input gradient.
 * @param[in]   num_sequences    number of sequences.
 * @param[in]   input_dim        input sequence dimension.
 * @param[in]   context_length   context length.
 * @param[in]   context_start    context start.
 *
 */
void hl_context_projection_backward_data(const real* out_grad,
                                         const int* sequence,
                                         real* input_grad,
                                         size_t num_sequences,
                                         size_t input_dim,
                                         size_t context_length,
                                         int context_start) {
  CHECK_NOTNULL(out_grad);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(input_grad);

  int block_size = 128;
  int blocks_x = num_sequences;
  int blocks_y = 1;
  dim3 threads(block_size, 1);
  dim3 grid(blocks_x, blocks_y);
  KeContextProjectionBackwardData<<< grid, threads, 0, STREAM_DEFAULT >>>
    (out_grad, sequence, input_grad, input_dim, context_length, context_start);
  CHECK_SYNC("hl_context_projection_backward_data failed");
}

template <>
void ContextProjectionBackwardData<DEVICE_TYPE_GPU>(const GpuMatrix& out_grad,
                                                    GpuMatrix& in_grad,
                                                    const GpuIVector& sequence,
                                                    size_t context_length,
                                                    int context_start) {
  hl_context_projection_backward_data(out_grad.getData(),
                                      sequence.getData(),
                                      in_grad.getData(),
                                      sequence.getSize() - 1,
                                      in_grad.getWidth(),
                                      context_length,
                                      context_start);
}

template<int THREADS_X, int THREADS_Y>
__global__ void KeContextProjectionBackwardWeight(const real* out_grad,
                                                  const int* sequence,
                                                  real* w_grad,
                                                  int num_sequences,
                                                  int w_dim,
                                                  int context_length,
                                                  int context_start,
                                                  int begin_pad) {
  __shared__ real sum_s[THREADS_Y][THREADS_X];
  int pad_of_block = (w_dim + THREADS_X - 1) / THREADS_X;
  const int idx = threadIdx.x;
  const int idy = threadIdx.y;
  int padId = blockIdx.x / pad_of_block;
  int weight_idx = idx + THREADS_X * (blockIdx.x % pad_of_block);
  int instanceId;
  real value = 0;
  real* output_r;

  sum_s[idy][idx] = 0.0f;
  if (weight_idx < w_dim) {
    for (int seqId = idy; seqId < num_sequences; seqId += THREADS_Y) {
      int seq_start = sequence[seqId];
      int seq_end = sequence[seqId+1];
      output_r = const_cast<real*>(out_grad)
                    + seq_start * w_dim * context_length;

      if (context_start < 0) {
        if (padId + context_start < 0) {
          instanceId = padId;
        } else {
          // begin_pad > 0;
          instanceId = (padId - begin_pad) +
            (seq_end - seq_start) - context_start;
        }
      } else {
        if (padId + (seq_end - seq_start) < context_start) {
          continue;
        } else {
          // begin_pad == 0;
          instanceId = padId + (seq_end - seq_start) - context_start;
        }
      }

      int outx = (instanceId - context_length) < 0 ?
                 instanceId : (context_length - 1);
      int outy = (instanceId - context_length) < 0 ?
                 0 : (instanceId - (context_length - 1));
      output_r += outy * w_dim * context_length + outx * w_dim;
      for (int j = outy; j < seq_end - seq_start; j++) {
        value += output_r[weight_idx];
        if (j - outy == outx) break;
        output_r += (context_length - 1) * w_dim;
      }
    }
    sum_s[idy][idx] = value;
  }
  __syncthreads();

  for (int stride = THREADS_Y/2; stride > 0; stride = stride/2) {
    if (idy < stride) {
      sum_s[idy][idx] += sum_s[idy + stride][idx];
    }
    __syncthreads();
  }
  __syncthreads();

  if (weight_idx < w_dim) {
    if (idy == 0) {
      w_grad[padId * w_dim + weight_idx] += sum_s[0][idx];
    }
  }
}

/**
 * @brief   Context projection backward weight.
 *
 * @param[in]   out_grad         output gradient.
 * @param[in]   sequence         sequence index.
 * @param[out]  w_grad           weight gradient.
 * @param[in]   num_sequences    number of sequences.
 * @param[in]   w_dim            input sequence dimension.
 * @param[in]   total_pad        number of extra timesteps.
 * @param[in]   context_length   context length.
 * @param[in]   context_start    context start.
 * @param[in]   begin_pad        number of extra timesteps added at the
 * beginning.
 *
 */
void hl_context_projection_backward_weight(const real* out_grad,
                                           const int* sequence,
                                           real* w_grad,
                                           size_t num_sequences,
                                           size_t w_dim,
                                           size_t total_pad,
                                           size_t context_length,
                                           int context_start,
                                           size_t begin_pad) {
  CHECK_NOTNULL(out_grad);
  CHECK_NOTNULL(sequence);
  CHECK_NOTNULL(w_grad);

  int threads_x = 32;
  int threads_y = 32;
  int blocks_x = total_pad * ((w_dim + threads_x - 1) / threads_x);
  dim3 threads(threads_x, threads_y);
  dim3 grid(blocks_x, 1);

  KeContextProjectionBackwardWeight<32, 32>
    <<< grid, threads, 0, STREAM_DEFAULT >>>
    (out_grad, sequence, w_grad, num_sequences, w_dim,
     context_length, context_start, begin_pad);
  CHECK_SYNC("hl_context_projection_backward_weight failed");
}

template <>
void ContextProjectionBackwardWeight<DEVICE_TYPE_GPU>(
        const GpuMatrix& out_grad,
        GpuMatrix& w_grad,
        const GpuIVector& seq_vec,
        size_t context_length,
        int context_start,
        size_t total_pad,
        size_t begin_pad) {
  hl_context_projection_backward_weight(out_grad.getData(),
                                        seq_vec.getData(),
                                        w_grad.getData(),
                                        seq_vec.getSize() - 1,
                                        w_grad.getWidth(),
                                        total_pad,
                                        context_length,
                                        context_start,
                                        begin_pad);
}

template <>
void ContextProjectionBackward<DEVICE_TYPE_GPU>(const GpuMatrix& out_grad,
                                                GpuMatrix& in_grad,
                                                GpuMatrix& w_grad,
                                                const GpuIVector& sequence,
                                                size_t context_length,
                                                int context_start,
                                                size_t begin_pad,
                                                bool is_padding,
                                                size_t total_pad) {
    if (in_grad) {
        ContextProjectionBackwardData<DEVICE_TYPE_GPU>(
                out_grad,
                in_grad,
                sequence,
                context_length,
                context_start);
    }
    if (is_padding && w_grad) {
        ContextProjectionBackwardWeight<DEVICE_TYPE_GPU>(
                out_grad,
                w_grad,
                sequence,
                context_length,
                context_start,
                total_pad,
                begin_pad);
  }
}

}  // namespace paddle
