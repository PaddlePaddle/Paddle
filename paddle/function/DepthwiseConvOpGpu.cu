#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "DepthwiseConvOp.h"
#include "paddle/math/BaseMatrix.h"

namespace paddle {

// CUDA kernel to compute the depthwise convolution forward pass
template <class T>
__global__ void ConvolutionDepthwiseForward(const int nthreads,
                                            const T* const inputData,
                                            const T* const filterData,
                                            const int batchSize,
                                            const int outputChannels,
                                            const int outputHeight,
                                            const int outputWidth,
                                            const int inputChannels,
                                            const int inputHeight,
                                            const int inputWidth,
                                            const int filterMultiplier,
                                            const int filterHeight,
                                            const int filterWidth,
                                            const int strideH,
                                            const int strideW,
                                            const int paddingH,
                                            const int paddingW,
                                            T* const outputData) {
  int index = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if (index < nthreads) {
    const int batch = index / outputChannels / outputHeight / outputWidth;
    const int c_out = (index / outputHeight / outputWidth) % outputChannels;
    const int h_out = (index / outputWidth) % outputHeight;
    const int w_out = index % outputWidth;

    const int c_in = c_out / filterMultiplier;
    const T* weight = filterData + c_out * filterHeight * filterWidth;
    T value = 0;
    const int h_in_start = -paddingH + h_out * strideH;
    const int w_in_start = -paddingW + w_out * strideW;
    const int h_in_end = -paddingH + h_out * strideH + filterHeight - 1;
    const int w_in_end = -paddingW + w_out * strideW + filterWidth - 1;
    if ((h_in_start >= 0) && (h_in_end < inputHeight) && (w_in_start >= 0) &&
        (w_in_end < inputWidth)) {
      for (int kh = 0; kh < filterHeight; ++kh) {
        for (int kw = 0; kw < filterWidth; ++kw) {
          const int h_in = -paddingH + h_out * strideH + kh;
          const int w_in = -paddingW + w_out * strideW + kw;
          const int offset =
              ((batch * inputChannels + c_in) * inputHeight + h_in) *
                  inputWidth +
              w_in;
          value += (*weight) * inputData[offset];
          ++weight;
        }
      }
    } else {
      for (int kh = 0; kh < filterHeight; ++kh) {
        for (int kw = 0; kw < filterWidth; ++kw) {
          const int h_in = -paddingH + h_out * strideH + kh;
          const int w_in = -paddingW + w_out * strideW + kw;
          if ((h_in >= 0) && (h_in < inputHeight) && (w_in >= 0) &&
              (w_in < inputWidth)) {
            const int offset =
                ((batch * inputChannels + c_in) * inputHeight + h_in) *
                    inputWidth +
                w_in;
            value += (*weight) * inputData[offset];
          }
          ++weight;
        }
      }
    }
    outputData[index] = value;
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t input.
template <class T>
__global__ void ConvolutionDepthwiseInputBackward(const int nthreads,
                                                  const T* const top_diff,
                                                  const T* const weight_data,
                                                  const int num,
                                                  const int outputChannels,
                                                  const int outputHeight,
                                                  const int outputWidth,
                                                  const int inputChannels,
                                                  const int inputHeight,
                                                  const int inputWidth,
                                                  const int filterMultiplier,
                                                  const int filterHeight,
                                                  const int filterWidth,
                                                  const int strideH,
                                                  const int strideW,
                                                  const int paddingH,
                                                  const int paddingW,
                                                  T* const bottom_diff) {
  int index = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    const int batch = index / inputChannels / inputHeight / inputWidth;
    const int c_in = (index / inputHeight / inputWidth) % inputChannels;
    const int h_in = (index / inputWidth) % inputHeight;
    const int w_in = index % inputWidth;

    const int c_out_start = c_in * filterMultiplier;

    int h_out_start = (h_in - filterHeight + paddingH + strideH) / strideH;
    h_out_start = 0 > h_out_start ? 0 : h_out_start;
    int h_out_end = (h_in + paddingH) / strideH;
    h_out_end = outputHeight - 1 < h_out_end ? outputHeight - 1 : h_out_end;
    int w_out_start = (w_in - filterWidth + paddingW + strideW) / strideW;
    w_out_start = 0 > w_out_start ? 0 : w_out_start;
    int w_out_end = (w_in + paddingW) / strideW;
    w_out_end = outputWidth - 1 < w_out_end ? outputWidth - 1 : w_out_end;

    T value = 0;

    for (int c_out = c_out_start; c_out < c_out_start + filterMultiplier;
         c_out++) {
      for (int h_out = h_out_start; h_out <= h_out_end; ++h_out) {
        const int filter_h = h_in + paddingH - h_out * strideH;
        for (int w_out = w_out_start; w_out <= w_out_end; ++w_out) {
          const int filter_w = w_in + paddingW - w_out * strideW;
          const int filter_offset = c_out * filterHeight * filterWidth +
                                    filter_h * filterWidth + filter_w;
          const int top_diff_offset =
              ((batch * outputChannels + c_out) * outputHeight + h_out) *
                  outputWidth +
              w_out;
          value += top_diff[top_diff_offset] * weight_data[filter_offset];
        }
      }
    }
    bottom_diff[index] += value;
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t filter.
template <class T>
__global__ void ConvolutionDepthwiseFilterBackward(const int num_i,
                                                   const int nthreads,
                                                   const T* const top_diff,
                                                   const T* const inputData,
                                                   const int num,
                                                   const int outputChannels,
                                                   const int outputHeight,
                                                   const int outputWidth,
                                                   const int inputChannels,
                                                   const int inputHeight,
                                                   const int inputWidth,
                                                   const int filterMultiplier,
                                                   const int filterHeight,
                                                   const int filterWidth,
                                                   const int strideH,
                                                   const int strideW,
                                                   const int paddingH,
                                                   const int paddingW,
                                                   T* const buffer_data) {
  int index = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    const int h_out = (index / outputWidth) % outputHeight;
    const int w_out = index % outputWidth;
    const int kh =
        (index / filterWidth / outputHeight / outputWidth) % filterHeight;
    const int kw = (index / outputHeight / outputWidth) % filterWidth;
    const int h_in = -paddingH + h_out * strideH + kh;
    const int w_in = -paddingW + w_out * strideW + kw;
    if ((h_in >= 0) && (h_in < inputHeight) && (w_in >= 0) &&
        (w_in < inputWidth)) {
      const int c_out =
          index / (filterHeight * filterWidth * outputHeight * outputWidth);
      const int c_in = c_out / filterMultiplier;
      const int batch = num_i;
      const int top_offset =
          ((batch * outputChannels + c_out) * outputHeight + h_out) *
              outputWidth +
          w_out;
      const int bottom_offset =
          ((batch * inputChannels + c_in) * inputHeight + h_in) * inputWidth +
          w_in;
      buffer_data[index] = top_diff[top_offset] * inputData[bottom_offset];
    } else {
      buffer_data[index] = 0;
    }
  }
}

template <class T>
class DepthwiseConvFunctor<DEVICE_TYPE_GPU, T> {
 public:
  void operator()(const T* inputData,
                  const T* filterData,
                  int batchSize,
                  int outputChannels,
                  int outputHeight,
                  int outputWidth,
                  int inputChannels,
                  int inputHeight,
                  int inputWidth,
                  int filterMultiplier,
                  int filterHeight,
                  int filterWidth,
                  int strideH,
                  int strideW,
                  int paddingH,
                  int paddingW,
                  T* outputData) {
    int outputSize = batchSize * outputChannels * outputHeight * outputWidth;

    size_t blocks = (outputSize + 1024 - 1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);

    ConvolutionDepthwiseForward<T><<<grid, threads, 0, STREAM_DEFAULT>>>(
        outputSize,
        inputData,
        filterData,
        batchSize,
        outputChannels,
        outputHeight,
        outputWidth,
        inputChannels,
        inputHeight,
        inputWidth,
        filterMultiplier,
        filterHeight,
        filterWidth,
        strideH,
        strideW,
        paddingH,
        paddingW,
        outputData);
  }
};

template <class T>
class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, T> {
 public:
  void operator()(const T* outputGrad,
                  const T* filterData,
                  int batchSize,
                  int outputChannels,
                  int outputHeight,
                  int outputWidth,
                  int inputChannels,
                  int inputHeight,
                  int inputWidth,
                  int filterMultiplier,
                  int filterHeight,
                  int filterWidth,
                  int strideH,
                  int strideW,
                  int paddingH,
                  int paddingW,
                  T* inputGrad) {
    int inputSize = batchSize * inputChannels * inputHeight * inputWidth;

    size_t blocks = (inputSize + 1024 - 1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);

    ConvolutionDepthwiseInputBackward<T>
        // NOLINT_NEXT_LINE(whitespace/operators)
        <<<grid, threads, 0, STREAM_DEFAULT>>>(inputSize,
                                               outputGrad,
                                               filterData,
                                               batchSize,
                                               outputChannels,
                                               outputHeight,
                                               outputWidth,
                                               inputChannels,
                                               inputHeight,
                                               inputWidth,
                                               filterMultiplier,
                                               filterHeight,
                                               filterWidth,
                                               strideH,
                                               strideW,
                                               paddingH,
                                               paddingW,
                                               inputGrad);
  }
};

template <class T>
class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, T> {
 public:
  void operator()(const T* outputGrad,
                  const T* inputData,
                  int batchSize,
                  int outputChannels,
                  int outputHeight,
                  int outputWidth,
                  int inputChannels,
                  int inputHeight,
                  int inputWidth,
                  int filterMultiplier,
                  int filterHeight,
                  int filterWidth,
                  int strideH,
                  int strideW,
                  int paddingH,
                  int paddingW,
                  T* colData,
                  T* filterGrad) {
    int colDataSize = outputChannels * filterHeight * filterWidth *
                      outputHeight * outputWidth;

    size_t blocks = (colDataSize + 1024 - 1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);
    BaseMatrix filterGradMatrix(outputChannels * filterHeight * filterWidth,
                                1,
                                filterGrad,
                                false,
                                true);

    for (int i = 0; i < batchSize; i++) {
      ConvolutionDepthwiseFilterBackward<
          T><<<grid, threads, 0, STREAM_DEFAULT>>>(i,
                                                   colDataSize,
                                                   outputGrad,
                                                   inputData,
                                                   batchSize,
                                                   outputChannels,
                                                   outputHeight,
                                                   outputWidth,
                                                   inputChannels,
                                                   inputHeight,
                                                   inputWidth,
                                                   filterMultiplier,
                                                   filterHeight,
                                                   filterWidth,
                                                   strideH,
                                                   strideW,
                                                   paddingH,
                                                   paddingW,
                                                   colData);
      int K = outputHeight * outputWidth;
      int M = colDataSize / K;

      BaseMatrix colMatrix(M, K, colData, false, true);
      filterGradMatrix.sumRows(colMatrix, (T)1.0, (T)1.0);
    }
  }
};

#ifdef PADDLE_TYPE_DOUBLE
template class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, double>;
template class DepthwiseConvFunctor<DEVICE_TYPE_GPU, double>;
template class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, double>;
#else
template class DepthwiseConvGradInputFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvFunctor<DEVICE_TYPE_GPU, float>;
template class DepthwiseConvGradFilterFunctor<DEVICE_TYPE_GPU, float>;
#endif

}  // namespace paddle
