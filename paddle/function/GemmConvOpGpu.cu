#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "ConvOp.h"
#include "GemmConvOp.h"

namespace paddle {

template<class T>
__global__
void im2col(const T* data_im, int numOuts, int height, int width,
            int blockH, int blockW,
            int strideH, int strideW,
            int paddingH, int paddingW,
            int height_col, int width_col,
            T* data_col) {
  int index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < numOuts) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * blockH * blockW;
    int h_in = h_out * strideH;
    int w_in = w_out * strideW;

    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    for (int i = 0; i < blockH; ++i) {
      for (int j = 0; j < blockW; ++j) {
        int rIdx = int(h_in+i);
        int cIdx = int(w_in+j);
        if ((rIdx-(int)paddingH) >= (int)height ||
            (rIdx-(int)paddingH) < 0 ||
            (cIdx-(int)paddingW) >= (int)width ||
            (cIdx-(int)paddingW) < 0) {
          *data_col = 0;
        } else {
          rIdx = rIdx + channel_in*height - paddingH;
          cIdx = cIdx - paddingW;
          *data_col = data_im[rIdx* width + cIdx];
        }
        data_col += height_col * width_col;
      }
    }
  }
}

template <class T>
class Im2ColFunctor<DEVICE_TYPE_GPU, T> {
public:
  void operator()(const T* imData,
                  int inputChannels,
                  int inputHeight,
                  int inputWidth,
                  int filterHeight,
                  int filterWidth,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth,
                  int outputHeight,
                  int outputWidth,
                  T* colData) {
    int numKernels = inputChannels * outputHeight * outputWidth;
    int blocks = (numKernels + 1024 -1) / 1024;
    int blockX = 512;
    int blockY = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);
    im2col<T><<< grid, threads, 0, STREAM_DEFAULT >>>
        (imData, numKernels, inputHeight, inputWidth, filterHeight, filterWidth,
         strideHeight, strideWidth, paddingHeight, paddingWidth,
         outputHeight, outputWidth, colData);
    CHECK_SYNC("Im2ColFunctor GPU failed");
  }
};

template<class T>
__global__
void col2im(size_t n, const T* data_col, size_t height,
            size_t width, size_t channels,
            size_t blockH, size_t blockW,
            size_t strideH, size_t strideW,
            size_t paddingH, size_t paddingW,
            size_t height_col, size_t width_col,
            T* data_im) {
  size_t index =
    (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < n) {
    T val = 0;
    int w = int(index % width);
    int h = int((index / width) % height);
    int c = int(index / (width * height));
    if ((w - (int)paddingW) >= 0 &&
        (w - (int)paddingW) < (width-2 * paddingW) &&
        (h - (int)paddingH) >= 0 &&
        (h - paddingH) < (height - 2 * paddingH)) {
      // compute the start and end of the output
      int w_col_start =
        (w < (int)blockW) ? 0 : (w - int(blockW)) / (int)strideW + 1;
      int w_col_end =
        min((int)(w / (int)strideW + 1), (int)(width_col));
      int h_col_start =
        (h < (int)blockH) ? 0 : (h - (int)blockH) / (int)strideH + 1;
      int h_col_end = min(int(h / strideH + 1), int(height_col));
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          // the col location: [c * width * height + h_out, w_out]
          int c_col = int(c * blockH* blockW) + \
            (h - h_col * (int)strideH) * (int)blockW +
            (w - w_col * (int)strideW);
          val += data_col[(c_col * height_col + h_col) * width_col + w_col];
        }
      }
      h -= paddingH;
      w -= paddingW;
      data_im[c*((width-2*paddingW) * (height-2*paddingH)) +
              h*(width-2*paddingW) + w] += val;
    }
  }
}

template <class T>
class Col2ImFunctor<DEVICE_TYPE_GPU, T> {
public:
  void operator()(const T* colData,
                  int inputChannels,
                  int inputHeight,
                  int inputWidth,
                  int filterHeight,
                  int filterWidth,
                  int strideHeight,
                  int strideWidth,
                  int paddingHeight,
                  int paddingWidth,
                  int outputHeight,
                  int outputWidth,
                  T* imData) {
    size_t numKernels = inputChannels * (inputHeight + 2*paddingHeight)
        * (inputWidth + 2*paddingWidth);

    size_t blocks = (numKernels + 1024 -1) / 1024;
    size_t blockX = 512;
    size_t blockY = (blocks+512-1)/512;
    dim3 threads(1024, 1);
    dim3 grid(blockX, blockY);

    // To avoid involving atomic operations, we will launch one kernel per
    // bottom dimension, and then in the kernel add up the top dimensions.
    col2im<T><<< grid, threads, 0, STREAM_DEFAULT >>>
             (numKernels,
              colData,
              inputHeight + 2*paddingHeight,
              inputWidth + 2*paddingWidth,
              inputChannels,
              filterHeight,
              filterWidth,
              strideHeight,
              strideWidth,
              paddingHeight,
              paddingWidth,
              outputHeight,
              outputWidth,
              imData);
    CHECK_SYNC("Col2ImFunctor GPU failed");
  }
};

template class Im2ColFunctor<DEVICE_TYPE_GPU, float>;
template class Im2ColFunctor<DEVICE_TYPE_GPU, double>;
template class Col2ImFunctor<DEVICE_TYPE_GPU, float>;
template class Col2ImFunctor<DEVICE_TYPE_GPU, double>;

}  // namespace paddle
