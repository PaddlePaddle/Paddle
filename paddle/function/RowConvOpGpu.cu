#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "RowConvOp.h"

namespace paddle {

template<int BLOCK_H, int BLOCK_W>
__global__ void KeRowConv(real* y, const real* x,  const real* w,
    const int* starts, const int height, const int width,
    const int numSeq, const int context) {

  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int blky = blockDim.y;
  const int gidx = blockIdx.x * blockDim.x;

  __shared__ real sw[BLOCK_H][BLOCK_W];

  for (int i = tidy; i < context; i += blky) {
    sw[i][tidx] = gidx + tidx < width ? w[i*width + gidx + tidx] : 0.0;
  }
  
  __syncthreads();

  for (int i = 0; i < numSeq; ++i) {
    const int start = starts[i];
    const int end = starts[i + 1];
    const int steps = end - start;
    for (int j = tidy; j < steps; j += blky) {
      real sum = 0;
      int off = (start + j) * width;
      for (int t = 0; t < context; ++t) {
        if ((start + j + t) < end) {
          int xoff = off + t * width;
          real xVal = gidx + tidx < width ? x[xoff + gidx + tidx] : 0.0;
          sum += sw[t][tidx] * xVal;
        }
      }
      if (gidx + tidx < width) {
        y[off + gidx + tidx] += sum;
      }
    }
  }
}

__global__ void KeRowConv2(real* y, const real* x,  const real* w,
    const int* starts, const int height, const int width,
    const int numSeq, const int context) {
  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int blky = blockDim.y;
  const int gidx = blockIdx.x * blockDim.x;

  for (int i = 0; i < numSeq; ++i) {
    const int start = starts[i];
    const int end = starts[i + 1];
    const int steps = end - start;
    for (int j = tidy; j < steps; j += blky) {
      int off = (start + j) * width;
      real sum = 0;
      for (int t = 0; t < context && (start + j + t) < end; ++t) {
        int xoff = off + t * width;
        real xd = gidx + tidx < width ? x[xoff + gidx + tidx] : 0.0;
        real wd = gidx + tidx < width ? w[t * width + gidx + tidx] : 0.0;
        sum += wd * xd;
      }
      if (gidx + tidx < width) {
        y[off + gidx + tidx] += sum;
      }
    }
  }
}



template <>
void RowConv<DEVICE_TYPE_GPU>(GpuMatrix& out,
                              const GpuMatrix& in,
                              const GpuMatrix& filter,
                              const GpuIVector& seq) {
  const size_t numSeq = seq.getSize() - 1;
  const size_t contextLength = filter.getHeight();
  const size_t height = in.getHeight();
  const size_t width = in.getWidth();

  real* y = out.getData();
  const real* x = in.getData();
  const real* w = filter.getData();
  const int* starts = seq.getData();

  dim3 dimBlock(32, 32);
  dim3 dimGrid(DIVUP(width, dimBlock.x), 1);

  if (contextLength <= 32) {
    KeRowConv<32, 32><<<dimGrid, dimBlock, 0, STREAM_DEFAULT>>>
      (y, x, w, starts, height, width, numSeq, contextLength);
  } else {
    KeRowConv2<<<dimGrid, dimBlock, 0, STREAM_DEFAULT>>>
      (y, x, w, starts, height, width, numSeq, contextLength);
  }
  CHECK_SYNC("RowConv");
}


template<int BLOCK_H, int BLOCK_W, int CONTEXT>
__global__ void KeRowConvBwWeight(real* dw, const real* x, const real* dy,
    const int* starts, const int height, const int width, const int numSeq,
    const int context) {

  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int blky = blockDim.y;
  const int gidx = blockIdx.x * blockDim.x;

  __shared__ real sh_x[BLOCK_W][BLOCK_H];
  __shared__ real sh_dy[BLOCK_W][BLOCK_H + CONTEXT - 1];
  __shared__ real sh_dw[CONTEXT][BLOCK_W];

  if (tidy < context) {
    sh_dw[tidy][tidx] = 0.0;
  }
  __syncthreads();

  for (int i = 0; i < numSeq; ++i) {
    const int start = starts[i];
    const int end = starts[i + 1];
    const int steps = end - start;
    const int size = ((steps + BLOCK_H - 1)/BLOCK_H) * BLOCK_H;
    for (int j = tidy; j < size; j += BLOCK_H) {
      int xoff = gidx + tidx;
      int yoff = start + j;

      // transpose
      sh_x[tidx][tidy] = (xoff < width && yoff < end) ? x[yoff * width + xoff] : 0.0;
      sh_dy[tidx][tidy + context - 1] = (xoff < width && yoff < end) ? dy[yoff * width + xoff] : 0.0;
      __syncthreads();
      if (tidy < (context - 1)) {
        yoff = yoff - context + 1;
        sh_dy[tidx][tidy] = (xoff < width && yoff >= start) ? dy[yoff * width + xoff] : 0.0;
      }
      __syncthreads();

      for (int t = 0; t < context; t++) {
        real val = sh_x[tidy][tidx] * sh_dy[tidy][tidx + context - 1 - t];
        __syncthreads();
        // warp size and blockDim.x is 32.
        val += __shfl_down(val, 16);
        val += __shfl_down(val, 8);
        val += __shfl_down(val, 4);
        val += __shfl_down(val, 2);
        val += __shfl_down(val, 1);
        __syncthreads();
        if (tidx == 0) {
          sh_dw[t][tidy] += val;
        }
        __syncthreads();
      }
    }
  }

  for (int t = tidy; (t < context) && ((gidx + tidx) < width); t += blky) {
    dw[t * width + gidx + tidx] += sh_dw[t][tidx];
  }
}

template<int BLOCK_H, int BLOCK_W>
__global__ void KeRowConvBwWeight2(real* dw, const real* x, const real* dy,
    const int* starts, const int height, const int width, const int numSeq,
    const int context) {

  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int gidx = blockIdx.x * blockDim.x;

  __shared__ real sh_x[BLOCK_H][BLOCK_W];
  __shared__ real sh_dy[BLOCK_H][BLOCK_W];

  for (int i = 0; i < numSeq; ++i) {
    const int start = starts[i];
    const int end = starts[i + 1];
    const int steps = end - start;

    const int size = ((steps + BLOCK_H - 1)/BLOCK_H) * BLOCK_H;
    for (int j = tidy; j < size; j += BLOCK_H) {
      int xoff = gidx + tidx;
      int yoff = start + j;

      // transpose
      sh_x[tidx][tidy] = (xoff < width && yoff < end) ? x[yoff * width + xoff] : 0.0;
      __syncthreads();

      for (int t = 0; t < context; t++) {
        sh_dy[tidx][tidy] = (xoff < width && (yoff - t) >= start && yoff - t < end) ? dy[(yoff - t) * width + xoff] : 0.0;
        __syncthreads();

        real val = sh_x[tidy][tidx] * sh_dy[tidy][tidx];
        __syncthreads();
        // warp size and blockDim.x is 32.
        val += __shfl_down(val, 16);
        val += __shfl_down(val, 8);
        val += __shfl_down(val, 4);
        val += __shfl_down(val, 2);
        val += __shfl_down(val, 1);
        __syncthreads();

        if (tidx == 0 && (gidx + tidy) < width) {
          dw[t*width + gidx + tidy] += val;
        }
      }
    }
  }
}

template<int BLOCK_H, int BLOCK_W>
__global__ void KeRowConvBwData(real* dx, const real* w, const real* dy,
    const int* starts, const int height, const int width, const int numSeq,
    const int context) {

  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int blky = blockDim.y;
  const int gidx = blockIdx.x * blockDim.x;

  __shared__ real sw[BLOCK_H][BLOCK_W];

  for (int i = tidy; i < context; i += blky) {
    sw[i][tidx] = gidx + tidx < width ? w[i*width + gidx + tidx] : 0.0;
  }
  
  __syncthreads();

  for (int i = 0; i < numSeq; ++i) {
    const int start = starts[i];
    const int end = starts[i + 1];
    const int steps = end - start;
    for (int j = tidy; j < steps; j += blky) {
      real sum = 0;
      int off = (start + j) * width;
      for (int t = 0; t < context && (j - t) >= 0; ++t) {
        int dyOff = off - t * width;
        real dyVal = gidx + tidx < width ? dy[dyOff + gidx + tidx] : 0.0;
        sum += sw[t][tidx] * dyVal;
      }
      if (gidx + tidx < width) {
        dx[off + gidx + tidx] += sum;
      }
    }
  }
}

__global__ void KeRowConvBwData2(real* dx, const real* w, const real* dy,
    const int* starts, const int height, const int width, const int numSeq,
    const int context) {

  const int tidx = threadIdx.x;
  const int tidy = threadIdx.y;
  const int blky = blockDim.y;
  const int gidx = blockIdx.x * blockDim.x;

  for (int i = 0; i < numSeq; ++i) {
    const int start = starts[i];
    const int end = starts[i + 1];
    const int steps = end - start;
    for (int j = tidy; j < steps; j += blky) {
      real sum = 0;
      int off = (start + j) * width;
      for (int t = 0; t < context && (j - t) >= 0; ++t) {
        int dyOff = off - t * width;
        real dyVal = gidx + tidx < width ? dy[dyOff + gidx + tidx] : 0.0;
        real wVal = gidx + tidx < width ? w[t * width + gidx + tidx] : 0.0;
        sum += wVal * dyVal;
      }
      if (gidx + tidx < width) {
        dx[off + gidx + tidx] += sum;
      }
    }
  }
}


template <>
void RowConvGrad<DEVICE_TYPE_GPU>(const GpuMatrix& outG,
                              const GpuMatrix& in,
                              const GpuMatrix& filter,
                              GpuMatrix& inG,
                              GpuMatrix& filterG,
                              const GpuIVector& seq) {
  const size_t numSeq = seq.getSize() - 1;
  const size_t contextLength = filter.getHeight();
  const size_t height = in.getHeight();
  const size_t width = in.getWidth();

  const real* dy = outG.getData();
  const real* x = in.getData();
  const real* w = filter.getData();
  const int* starts = seq.getData();

  if (filterG) {
    dim3 dimBlock(32, 32);
    dim3 dimGrid(DIVUP(width, dimBlock.x), 1);
    real* dw = filterG.getData();
    if (contextLength <= 32) { 
      KeRowConvBwWeight<32, 32, 32>
        <<<dimGrid, dimBlock, 0, STREAM_DEFAULT>>>
        (dw, x, dy, starts, height, width, numSeq, contextLength);
    } else {
      KeRowConvBwWeight2<32, 32>
        <<<dimGrid, dimBlock, 0, STREAM_DEFAULT>>>
        (dw, x, dy, starts, height, width, numSeq, contextLength);
    }
  }

  if (inG) {
    real* dx = inG.getData();
    dim3 dimBlock2(32, 32);
    dim3 dimGrid2(DIVUP(width, dimBlock2.x), 1);
    if (contextLength <= 64) {
      KeRowConvBwData<32, 64>
        <<<dimGrid2, dimBlock2, 0, STREAM_DEFAULT>>>
        (dx, w, dy, starts, height, width, numSeq, contextLength);
    } else {
      KeRowConvBwData2
        <<<dimGrid2, dimBlock2, 0, STREAM_DEFAULT>>>
        (dx, w, dy, starts, height, width, numSeq, contextLength);
    }
  }

  CHECK_SYNC("RowConvGrad");
}

}  // namespace paddle
