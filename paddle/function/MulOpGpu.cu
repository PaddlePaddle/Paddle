/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "MulOp.h"
#include "paddle/math/Matrix.h"
#include "paddle/math/SparseMatrix.h"

namespace paddle {
/// dense matrix (+)= dense matrix * dense matrix
template <>
void MulOp<DEVICE_TYPE_GPU>(GpuMatrix& out,
                            const GpuMatrix& a,
                            const GpuMatrix& b,
                            real scaleAB,
                            real scaleT,
                            bool aTrans,
                            bool bTrans) {
  CHECK(a.useGpu_ && b.useGpu_) << "matrix device type not match";
  hl_matrix_mul(const_cast<real*>(a.getData()),
                !aTrans ? HPPL_OP_N : HPPL_OP_T,
                const_cast<real*>(b.getData()),
                !bTrans ? HPPL_OP_N : HPPL_OP_T,
                const_cast<real*>(out.getData()),
                out.getHeight(),
                out.getWidth(),
                !aTrans ? a.getWidth() : a.getHeight(),
                scaleAB,
                scaleT,
                a.getStride(),
                b.getStride(),
                out.getStride());
}

/// dense matrix (+)= sparse matrix * dense matrix
template <>
void MulOp<DEVICE_TYPE_GPU>(GpuMatrix& out,
                            const GpuSparseMatrix& a,
                            const GpuMatrix& b,
                            real scaleAB,
                            real scaleT,
                            bool aTrans,
                            bool bTrans) {
  CHECK(out.isContiguous());
  CHECK(b.isContiguous());
  CHECK(a.useGpu_ && b.useGpu_) << "matrix device type not match";
  hl_matrix_csr_mul_dense(a.sMatrix_.get(),
                          aTrans ? HPPL_OP_T : HPPL_OP_N,
                          const_cast<real*>(b.getData()),
                          HPPL_OP_N,
                          const_cast<real*>(out.getData()),
                          out.getHeight(),
                          out.getWidth(),
                          b.getHeight(),
                          scaleAB,
                          scaleT);
}

/// dense matrix (+)= dense matrix * sparse matrix
template <>
void MulOp<DEVICE_TYPE_GPU>(GpuMatrix& out,
                            const GpuMatrix& a,
                            const GpuSparseMatrix& b,
                            real scaleAB,
                            real scaleT,
                            bool aTrans,
                            bool bTrans) {
  CHECK(out.isContiguous());
  CHECK(a.isContiguous());
  CHECK(a.useGpu_ && b.useGpu_) << "matrix device type not match";

  if (b.format_ == SPARSE_CSC) {
    hl_matrix_dense_mul_csc(const_cast<real*>(a.getData()),
                            HPPL_OP_N,
                            b.sMatrix_.get(),
                            bTrans ? HPPL_OP_T : HPPL_OP_N,
                            const_cast<real*>(out.getData()),
                            out.getHeight(),
                            out.getWidth(),
                            a.getWidth(),
                            scaleAB,
                            scaleT);
  } else {
    hl_matrix_dense_mul_csr(const_cast<real*>(a.getData()),
                            HPPL_OP_N,
                            b.sMatrix_.get(),
                            bTrans ? HPPL_OP_T : HPPL_OP_N,
                            const_cast<real*>(out.getData()),
                            out.getHeight(),
                            out.getWidth(),
                            a.getWidth(),
                            scaleAB,
                            scaleT);
  }
}

/// sparse matrix (+)= dense matrix * dense matrix
template <>
void MulOp<DEVICE_TYPE_GPU>(GpuSparseMatrix& out,
                            const GpuMatrix& a,
                            const GpuMatrix& b,
                            real scaleAB,
                            real scaleT,
                            bool aTrans,
                            bool bTrans) {
  CHECK(a.useGpu_ && b.useGpu_) << "matrix device type not match";
  hl_sparse_matrix_mul(const_cast<real*>(a.getData()),
                       aTrans ? HPPL_OP_T : HPPL_OP_N,
                       const_cast<real*>(b.getData()),
                       bTrans ? HPPL_OP_T : HPPL_OP_N,
                       out.sMatrix_.get(),
                       out.getHeight(),
                       out.getWidth(),
                       !bTrans ? b.getHeight() : b.getWidth(),
                       scaleAB,
                       scaleT);
}

}  // namespace paddle
