#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hl_base.h"
#include "PadOp.h"

namespace paddle {

__global__ void KePad(real* outputs, const real* inputs,
                      int inC, int inH, int inW,
                      int padc, int padh, int padw,
                      int outC, int outH, int outW, int nthreads) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nthreads) {
    const int w = idx % inW;
    const int h = (idx / inW) % inH;
    const int c = (idx / inW / inH) % inC;
    const int n = idx / inW / inH / inC;

    const int off = ((n * outC + c + padc) * outH + h + padh) * outW + padw + w;
    outputs[off] = inputs[idx];
  }
}

template <>
void Pad<DEVICE_TYPE_GPU>(real* outputs,
                          const real* inputs,
                          const int num,
                          const int inC,
                          const int inH,
                          const int inW,
                          const PadConf& pad) {
  size_t nth = num * inC * inH * inW;
  int blockSize = 1024;
  int gridSize = (nth + 1024 - 1) / 1024;
  int cstart = pad.channelStart, cend = pad.channelEnd;
  int hstart = pad.heightStart, hend = pad.heightEnd;
  int wstart = pad.widthStart, wend = pad.widthEnd;
  int outC = inC + cstart + cend;
  int outH = inH + hstart + hend;
  int outW = inW + wstart + wend;
  KePad<<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (outputs, inputs, inC, inH, inW, cstart, hstart, wstart,
     outC, outH, outW, nth);
  CHECK_SYNC("Pad");
}

__global__ void KePadDiff(real* inGrad, const real* outGrad,
                          int inC, int inH, int inW,
                          int padc, int padh, int padw,
                          int outC, int outH, int outW, int nthreads) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nthreads) {
    const int w = idx % inW;
    const int h = (idx / inW) % inH;
    const int c = (idx / inW / inH) % inC;
    const int n = idx / inW / inH / inC;

    const int off = ((n * outC + c + padc) * outH + h + padh) * outW + padw + w;
    inGrad[idx] += outGrad[off];
  }
}

template <>
void PadGrad<DEVICE_TYPE_GPU>(real* inGrad,
                              const real* outGrad,
                              const int num,
                              const int inC,
                              const int inH,
                              const int inW,
                              const PadConf& pad) {
  int nth = num * inC * inH * inW;
  int blockSize = 1024;
  int gridSize = (nth + 1024 - 1) / 1024;
  int cstart = pad.channelStart, cend = pad.channelEnd;
  int hstart = pad.heightStart, hend = pad.heightEnd;
  int wstart = pad.widthStart, wend = pad.widthEnd;
  int outC = inC + cstart + cend;
  int outH = inH + hstart + hend;
  int outW = inW + wstart + wend;
  KePadDiff <<<gridSize, blockSize, 0, STREAM_DEFAULT>>>
    (inGrad, outGrad, inC, inH, inW, cstart, hstart, wstart,
     outC, outH, outW, nth);
  CHECK_SYNC("PadGrad");
}

}  // namespace paddle
