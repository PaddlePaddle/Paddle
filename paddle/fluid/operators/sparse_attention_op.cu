#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <math.h>
#include <limits>
#include <string>
#include <vector>
#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/framework/op_registry.h"
#if defined(PADDLE_WITH_CUDA)
#include "paddle/fluid/platform/dynload/hipsparse.h"
#endif

namespace ops = paddle::operators;
namespace plf = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
__forceinline__ __device__ T CudaShuffleXorSync(unsigned mask, T val,
                                                int width = warpSize) {
  return __shfl_xor_sync(mask, val, width);
}

template <typename T, int batch_size, int warp_size>
__device__ __forceinline__ void WarpReduceSum(T* sum) {
#pragma unroll
  for (int offset = warp_size / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < batch_size; ++i) {
      T sum_val = CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = sum[i] + sum_val;
    }
  }
}

template <typename T, int batch_size, int warp_size>
__device__ __forceinline__ void WarpReduceMax(T* sum) {
#pragma unroll
  for (int offset = warp_size / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < batch_size; ++i) {
      T max_val = CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = max(sum[i], max_val);
    }
  }
}

template <typename T, int BlockSize, int BlockNnzMax>
__global__ void BlockSparseSoftmaxForward(T* softmax, const T* src, T scale,
                                          const T* kp_mask, const T* attn_mask,
                                          const int* layout_rowptr,
                                          const int* layout_colindex,
                                          int num_rows) {
  // current thread related info
  const int WarpSize = 32;
  const int cur_row = blockIdx.x * blockDim.y + threadIdx.y;
  if (cur_row < num_rows) {
    const int cur_block_row = cur_row / BlockSize;
    const int cur_block_nnz =
        layout_rowptr[cur_block_row + 1] - layout_rowptr[cur_block_row];

    T srcdata[(BlockSize * BlockNnzMax + WarpSize - 1) / WarpSize] = {0};
    T attndata[(BlockSize * BlockNnzMax + WarpSize - 1) / WarpSize] = {0};

    // read tensor data, attn mask
    const int iter = (cur_block_nnz + WarpSize - 1) / WarpSize;
    const T* srcptr = src + layout_rowptr[cur_block_row];

    const T* attnptr = (attn_mask == nullptr)
                           ? nullptr
                           : (attn_mask + cur_block_row * num_rows);
    // the coloumn start index in current row
    const int* colindex = layout_colindex + layout_rowptr[cur_block_row];
    for (int j = 0; j < iter; j++) {
      int cur_block_col = j * WarpSize + threadIdx.x;
      int cur_reg_index = j;
      if (cur_block_col < cur_block_nnz) {
        // read kp mask
        T cur_kp_mask;
        if ((kp_mask != nullptr) &&
            std::abs(kp_mask[colindex[cur_block_col]]) <
                std::numeric_limits<T>::epsilon()) {
          cur_kp_mask = -std::numeric_limits<T>::infinity();
        } else {
          cur_kp_mask = 0;
        }
        // do mask operation
        if ((attnptr != nullptr) &&
            std::abs(attnptr[colindex[cur_block_col]]) <
                std::numeric_limits<T>::epsilon()) {
          srcdata[cur_reg_index] =
              -std::numeric_limits<T>::infinity() * scale + cur_kp_mask;
        } else {
          srcdata[cur_reg_index] = scale * srcptr[cur_block_col] + cur_kp_mask;
        }
      } else {
        srcdata[cur_reg_index] = -std::numeric_limits<T>::infinity();
      }
    }

    // max value
    T max_value = srcdata[0];
    const int kIteration =
        (cur_block_nnz * BlockSize + WarpSize - 1) / WarpSize;
#pragma unroll
    for (int it = 1; it < kIteration; ++it) {
      max_value = (max_value > srcdata[it]) ? max_value : srcdata[it];
    }
    WarpReduceMax<T, 1, WarpSize>(&max_value);

    // exp sum
    T sum = 0;
#pragma unroll
    for (int it = 0; it < kIteration; ++it) {
      srcdata[it] = std::exp(srcdata[it] - max_value);
      sum += srcdata[it];
    }
    WarpReduceSum<T, 1, WarpSize>(&sum);

    // compute softmax and write out
    T* softmaxptr = softmax + layout_rowptr[cur_block_row];
    for (int j = 0; j < iter; j++) {
      int cur_block_col = j * WarpSize + threadIdx.x;
      int cur_reg_index = j;
      if (cur_block_col < cur_block_nnz) {
        softmaxptr[cur_block_col] = srcdata[cur_reg_index] / sum;
      }
    }
  }
}

template <typename T, int BlockSize, int BlockNnzMax>
__global__ void BlockSparseSoftmaxBackward(T* dst, const T* grad, const T* src,
                                           T scale, const int* layout_rowptr,
                                           const int* layout_colindex,
                                           int num_rows) {
  // current thread related info
  const int WarpSize = 32;
  const int cur_row = blockIdx.x * blockDim.y + threadIdx.y;
  if (cur_row < num_rows) {
    const int cur_block_row = cur_row / BlockSize;
    const int cur_block_nnz =
        layout_rowptr[cur_block_row + 1] - layout_rowptr[cur_block_row];

    T srcdata[(BlockSize * BlockNnzMax + WarpSize - 1) / WarpSize];
    T graddata[(BlockSize * BlockNnzMax + WarpSize - 1) / WarpSize];

    // read tensor data, attn mask
    const int iter = (cur_block_nnz + WarpSize - 1) / WarpSize;
    const T* srcptr = src + layout_rowptr[cur_block_row];
    const T* gradptr = grad + layout_rowptr[cur_block_row];
    for (int j = 0; j < iter; j++) {
      int cur_block_col = j * WarpSize + threadIdx.x;
      int cur_reg_index = j;
      if (cur_block_col < cur_block_nnz) {
        srcdata[cur_reg_index] = srcptr[cur_block_col];
        graddata[cur_reg_index] = gradptr[cur_block_col];
      } else {
        srcdata[cur_reg_index] = 0;
        graddata[cur_reg_index] = 0;
      }
    }

    T sum = 0;
    const int kIteration =
        (cur_block_nnz * BlockSize + WarpSize - 1) / WarpSize;
#pragma unroll
    for (int it = 0; it < kIteration; ++it) {
      sum += srcdata[it] * graddata[it];
    }
    WarpReduceSum<T, 1, WarpSize>(&sum);

    // compute softmax and write out
    T* dstptr = dst + layout_rowptr[cur_block_row];
    for (int j = 0; j < iter; j++) {
      int cur_block_col = j * WarpSize + threadIdx.x;
      int cur_reg_index = j;
      if (cur_block_col < cur_block_nnz) {
        dstptr[cur_block_col] =
            scale * srcdata[cur_reg_index] * (graddata[cur_reg_index] - sum);
      }
    }
  }
}

using Tensor = framework::Tensor;
/*
input: sparse C in CSR format (num_rows,num_rows)
output: sparse C after softmax operation
*/
template <typename DeviceContext, typename T>
void SparseSoftmaxForward(const platform::CUDADeviceContext& ctx,
                          const Tensor* offset, const Tensor* columns,
                          Tensor* input, Tensor* output, const int blocksize,
                          const int num_rows, const int num_cols,
                          const Tensor* key_padding_mask,
                          const Tensor* attn_mask) {
  const int* offset_data = offset->data<int>();
  const int* columns_data = columns->data<int>();
  T* input_data = input->data<T>();
  T* output_data = output->data<T>();
  // Add mask
  const T* key_padding_mask_data =
      (key_padding_mask != nullptr) ? key_padding_mask->data<T>() : nullptr;
  const T* attn_mask_data =
      (attn_mask != nullptr) ? attn_mask->data<T>() : nullptr;

  const int block_size = 1;
  dim3 blocks(32, 4, 1);
  int grid = (num_rows * block_size + 3) / 4;
  T scaling = static_cast<T>(1.0) / sqrt(static_cast<T>(num_cols));

  if (num_cols <= 4) {
    BlockSparseSoftmaxForward<T, block_size, 4><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 4 && num_cols <= 8) {
    BlockSparseSoftmaxForward<T, block_size, 8><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 8 && num_cols <= 16) {
    BlockSparseSoftmaxForward<T, block_size, 16><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 16 && num_cols <= 32) {
    BlockSparseSoftmaxForward<T, block_size, 32><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 32 && num_cols <= 64) {
    BlockSparseSoftmaxForward<T, block_size, 64><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 64 && num_cols <= 128) {
    BlockSparseSoftmaxForward<T, block_size, 128><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 128 && num_cols <= 256) {
    BlockSparseSoftmaxForward<T, block_size, 256><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else if (num_cols > 256 && num_cols <= 512) {
    BlockSparseSoftmaxForward<T, block_size, 512><<<grid, blocks>>>(
        output_data, input_data, scaling, key_padding_mask_data, attn_mask_data,
        offset_data, columns_data, num_rows);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The head_dim of query in sparse_attention op should less or equal "
        "512"));
  }
}

template <typename DeviceContext, typename T>
void SparseSoftmaxBackward(const platform::CUDADeviceContext& ctx,
                           const Tensor* offset, const Tensor* columns,
                           Tensor* dx, const Tensor* dout, const Tensor* out,
                           const int blocksize, const int num_rows,
                           const int num_cols) {
  const int* offset_data = offset->data<int>();
  const int* columns_data = columns->data<int>();
  T* dx_data = dx->data<T>();
  const T* dout_data = dout->data<T>();
  const T* out_data = out->data<T>();

  const int block_size = 1;
  dim3 blocks(32, 4, 1);
  int grid = (num_rows * block_size + 3) / 4;
  T scaling = static_cast<T>(1.0) / sqrt(static_cast<T>(num_cols));

  if (num_cols <= 4) {
    BlockSparseSoftmaxBackward<T, block_size, 4><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 4 && num_cols <= 8) {
    BlockSparseSoftmaxBackward<T, block_size, 8><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 8 && num_cols <= 16) {
    BlockSparseSoftmaxBackward<T, block_size, 16><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 16 && num_cols <= 32) {
    BlockSparseSoftmaxBackward<T, block_size, 32><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 32 && num_cols <= 64) {
    BlockSparseSoftmaxBackward<T, block_size, 64><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 64 && num_cols <= 128) {
    BlockSparseSoftmaxBackward<T, block_size, 128><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 128 && num_cols <= 256) {
    BlockSparseSoftmaxBackward<T, block_size, 256><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else if (num_cols > 256 && num_cols <= 512) {
    BlockSparseSoftmaxBackward<T, block_size, 512><<<grid, blocks>>>(
        dx_data, dout_data, out_data, scaling, offset_data, columns_data,
        num_rows);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The head_dim of query in sparse_attention op should less or equal "
        "512"));
  }
}

using VarType = framework::proto::VarType;
inline hipDataType GetGpuType(const VarType::Type data_type) {
  if (data_type == VarType::FP32) {
    return HIP_R_32F;
  } else if (data_type == VarType::FP64) {
    return HIP_R_64F;
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "Not support tensor type in sparse_attention OP: %s",
        framework::DataTypeToString(data_type)));
  }
}

inline hipsparseOperation_t GetTransposeOperation(const bool transpose) {
  if (transpose) {
    return HIPSPARSE_OPERATION_TRANSPOSE;
  } else {
    return HIPSPARSE_OPERATION_NON_TRANSPOSE;
  }
}

void CusparseDestroy(hipsparseDnMatDescr_t* dn_mat_first,
                     hipsparseDnMatDescr_t* dn_mat_second,
                     hipsparseSpMatDescr_t* sp_mat) {
  platform::dynload::hipsparseDestroyDnMat(*dn_mat_first);
  platform::dynload::hipsparseDestroyDnMat(*dn_mat_second);
  platform::dynload::hipsparseDestroySpMat(*sp_mat);
}

/*
input: dense A (num_rows,num_cols), dense B (num_rows,num_cols)
output: sparse C in CSR format (num_rows,num_rows)
*/
template <typename DeviceContext, typename T>
void DotSdd(const platform::CUDADeviceContext& ctx, const Tensor* a,
            const Tensor* b, const Tensor* c_offset, const Tensor* c_columns,
            Tensor* c_value, const int num_rows, const int num_cols,
            const bool a_transpose, const bool b_transpose) {
  const T* a_data = a->data<T>();
  const T* b_data = b->data<T>();
  const int* c_offset_data = c_offset->data<int>();
  const int* c_columns_data = c_columns->data<int>();
  T* c_value_data = c_value->data<T>();

  hipDataType gpu_type =
      GetGpuType(framework::TransToProtoVarType(c_value->dtype()));
  hipsparseHandle_t handle = nullptr;
  hipsparseDnMatDescr_t mat_a, mat_b;
  hipsparseSpMatDescr_t mat_c;
  platform::dynload::hipsparseCreate(&handle);

  // Create dense matrix A
  platform::dynload::hipsparseCreateDnMat(&mat_a, num_rows, num_cols, num_cols,
                                         const_cast<T*>(a_data), gpu_type,
                                         HIPSPARSE_ORDER_ROW);
  // Create dense matrix B
  platform::dynload::hipsparseCreateDnMat(&mat_b, num_rows, num_cols, num_cols,
                                         const_cast<T*>(b_data), gpu_type,
                                         HIPSPARSE_ORDER_ROW);
  // Create sparse matrix C in CSR format
  int c_nnz = c_columns->dims()[1];
  platform::dynload::hipsparseCreateCsr(
      &mat_c, num_rows, num_rows, c_nnz, const_cast<int*>(c_offset_data),
      const_cast<int*>(c_columns_data), c_value_data, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, gpu_type);

  T alpha = 1;
  T beta = 0;

  size_t buffer_size = 0;
  platform::dynload::hipsparseSDDMM_bufferSize(
      handle, GetTransposeOperation(a_transpose),
      GetTransposeOperation(b_transpose), &alpha, mat_a, mat_b, &beta, mat_c,
      gpu_type, HIPSPARSE_SDDMM_ALG_DEFAULT, &buffer_size);
  auto d_buffer_ptr = paddle::memory::Alloc(ctx, buffer_size);
  void* d_buffer = static_cast<void*>(d_buffer_ptr->ptr());

  platform::dynload::hipsparseSDDMM(handle, GetTransposeOperation(a_transpose),
                                   GetTransposeOperation(b_transpose), &alpha,
                                   mat_a, mat_b, &beta, mat_c, gpu_type,
                                   HIPSPARSE_SDDMM_ALG_DEFAULT, d_buffer);

  CusparseDestroy(&mat_a, &mat_b, &mat_c);
  platform::dynload::hipsparseDestroy(handle);
}

/*
input: sparse A in CSR format (num_rows,num_rows), dense B (num_rows,num_cols)
output: dense C (num_rows,num_cols)
*/
template <typename DeviceContext, typename T>
void DotDsd(const platform::CUDADeviceContext& ctx, const Tensor* a_offset,
            const Tensor* a_columns, const Tensor* a_value, const Tensor* b,
            Tensor* c, const int num_rows, const int num_cols,
            const bool a_transpose, const bool b_transpose) {
  const int* a_offset_data = a_offset->data<int>();
  const int* a_columns_data = a_columns->data<int>();
  const T* a_value_data = a_value->data<T>();
  const T* b_data = b->data<T>();
  T* c_data = c->data<T>();

  hipDataType gpu_type =
      GetGpuType(framework::TransToProtoVarType(c->dtype()));
  hipsparseHandle_t handle = nullptr;
  hipsparseSpMatDescr_t mat_a;
  hipsparseDnMatDescr_t mat_b, mat_c;
  platform::dynload::hipsparseCreate(&handle);

  // Create sparse matrix A in CSR format
  int a_nnz = a_columns->dims()[1];
  platform::dynload::hipsparseCreateCsr(
      &mat_a, num_rows, num_rows, a_nnz, const_cast<int*>(a_offset_data),
      const_cast<int*>(a_columns_data), const_cast<T*>(a_value_data),
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
      gpu_type);

  // Create dense matrix B
  platform::dynload::hipsparseCreateDnMat(&mat_b, num_rows, num_cols, num_cols,
                                         const_cast<T*>(b_data), gpu_type,
                                         HIPSPARSE_ORDER_ROW);
  // Create dense matrix C
  platform::dynload::hipsparseCreateDnMat(&mat_c, num_rows, num_cols, num_cols,
                                         c_data, gpu_type, HIPSPARSE_ORDER_ROW);

  T alpha = 1;
  T beta = 0;

  size_t buffer_size = 0;
  // allocate an external buffer if needed
  platform::dynload::hipsparseSpMM_bufferSize(
      handle, GetTransposeOperation(a_transpose),
      GetTransposeOperation(b_transpose), &alpha, mat_a, mat_b, &beta, mat_c,
      gpu_type, HIPSPARSE_SPMM_ALG_DEFAULT, &buffer_size);
  auto d_buffer_ptr = paddle::memory::Alloc(ctx, buffer_size);
  void* d_buffer = static_cast<void*>(d_buffer_ptr->ptr());

  platform::dynload::hipsparseSpMM(handle, GetTransposeOperation(a_transpose),
                                  GetTransposeOperation(b_transpose), &alpha,
                                  mat_a, mat_b, &beta, mat_c, gpu_type,
                                  HIPSPARSE_SPMM_ALG_DEFAULT, d_buffer);

  CusparseDestroy(&mat_b, &mat_c, &mat_a);
  platform::dynload::hipsparseDestroy(handle);
}

std::vector<Tensor> GetSplitTensor(Tensor* input) {
  auto dims = input->dims();
  int batch_size = dims[0];
  int num_heads = dims[1];
  std::vector<int> new_dims(dims.size() - 1);
  new_dims[0] = batch_size * num_heads;
  for (int i = 1; i < new_dims.size(); i++) {
    new_dims[i] = dims[i + 1];
  }
  input->Resize(phi::make_ddim(new_dims));
  return input->Split(1, 0);
}

template <typename DeviceContext, typename T>
class SparseAttentionCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto query = *ctx.Input<Tensor>("Q");
    auto key = *ctx.Input<Tensor>("K");
    auto value = *ctx.Input<Tensor>("V");
    auto offset = *ctx.Input<Tensor>("Offset");
    auto columns = *ctx.Input<Tensor>("Columns");
    auto output_ptr = ctx.Output<Tensor>("Out");
    output_ptr->mutable_data<T>(ctx.GetPlace());
    auto sparse_dot_sdd_ptr = ctx.Output<Tensor>("SparseDotSdd");
    sparse_dot_sdd_ptr->mutable_data<T>(ctx.GetPlace());
    auto softmax_ptr = ctx.Output<Tensor>("Softmax");
    softmax_ptr->mutable_data<T>(ctx.GetPlace());
    // add Mask
    auto* key_padding_mask = ctx.HasInput("KeyPaddingMask")
                                 ? ctx.Input<Tensor>("KeyPaddingMask")
                                 : nullptr;
    auto* attn_mask =
        ctx.HasInput("AttnMask") ? ctx.Input<Tensor>("AttnMask") : nullptr;

    auto output = *output_ptr;
    auto result_sdd = *sparse_dot_sdd_ptr;
    auto result_softmax = *softmax_ptr;

    auto query_dims = query.dims();
    int batch_size = query_dims[0];
    int num_heads = query_dims[1];
    int M = query_dims[2];
    int N = query_dims[3];

    std::vector<Tensor> query_lists = GetSplitTensor(&query);
    std::vector<Tensor> key_lists = GetSplitTensor(&key);
    std::vector<Tensor> value_lists = GetSplitTensor(&value);
    std::vector<Tensor> offset_lists = GetSplitTensor(&offset);
    std::vector<Tensor> columns_lists = GetSplitTensor(&columns);
    std::vector<Tensor> result_sdd_lists = GetSplitTensor(&result_sdd);
    std::vector<Tensor> result_softmax_lists = GetSplitTensor(&result_softmax);
    std::vector<Tensor> output_lists = GetSplitTensor(&output);

    const auto& dev_ctx = ctx.cuda_device_context();
    const int iter_num = batch_size * num_heads;
    for (int i = 0; i < iter_num; i++) {
      DotSdd<DeviceContext, T>(dev_ctx, &query_lists[i], &key_lists[i],
                               &offset_lists[i], &columns_lists[i],
                               &result_sdd_lists[i], M, N, false, true);

      if (key_padding_mask != nullptr && attn_mask != nullptr) {
        SparseSoftmaxForward<DeviceContext, T>(
            dev_ctx, &offset_lists[i], &columns_lists[i], &result_sdd_lists[i],
            &result_softmax_lists[i], 1, M, N,
            key_padding_mask + (i / num_heads) * M, attn_mask);
      } else if (key_padding_mask != nullptr && attn_mask == nullptr) {
        SparseSoftmaxForward<DeviceContext, T>(
            dev_ctx, &offset_lists[i], &columns_lists[i], &result_sdd_lists[i],
            &result_softmax_lists[i], 1, M, N,
            key_padding_mask + (i / num_heads) * M, nullptr);
      } else if (key_padding_mask == nullptr && attn_mask != nullptr) {
        SparseSoftmaxForward<DeviceContext, T>(
            dev_ctx, &offset_lists[i], &columns_lists[i], &result_sdd_lists[i],
            &result_softmax_lists[i], 1, M, N, nullptr, attn_mask);
      } else {
        SparseSoftmaxForward<DeviceContext, T>(
            dev_ctx, &offset_lists[i], &columns_lists[i], &result_sdd_lists[i],
            &result_softmax_lists[i], 1, M, N, nullptr, nullptr);
      }

      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &result_softmax_lists[i], &value_lists[i],
                               &output_lists[i], M, N, false, false);
    }
  }
};

template <typename DeviceContext, typename T>
class SparseAttentionGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto query = *ctx.Input<Tensor>("Q");
    auto key = *ctx.Input<Tensor>("K");
    auto value = *ctx.Input<Tensor>("V");
    auto offset = *ctx.Input<Tensor>("Offset");
    auto columns = *ctx.Input<Tensor>("Columns");
    auto sparse_dot_sdd = *ctx.Input<Tensor>("SparseDotSdd");
    auto softmax = *ctx.Input<Tensor>("Softmax");
    auto dout = *ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dquery_ptr = ctx.Output<Tensor>(framework::GradVarName("Q"));
    auto* dkey_ptr = ctx.Output<Tensor>(framework::GradVarName("K"));
    auto* dvalue_ptr = ctx.Output<Tensor>(framework::GradVarName("V"));
    dquery_ptr->mutable_data<T>(ctx.GetPlace());
    dkey_ptr->mutable_data<T>(ctx.GetPlace());
    dvalue_ptr->mutable_data<T>(ctx.GetPlace());
    auto dquery = *dquery_ptr;
    auto dkey = *dkey_ptr;
    auto dvalue = *dvalue_ptr;

    auto query_dims = query.dims();
    int batch_size = query_dims[0];
    int num_heads = query_dims[1];
    int M = query_dims[2];
    int N = query_dims[3];

    std::vector<Tensor> query_lists = GetSplitTensor(&query);
    std::vector<Tensor> key_lists = GetSplitTensor(&key);
    std::vector<Tensor> value_lists = GetSplitTensor(&value);
    std::vector<Tensor> offset_lists = GetSplitTensor(&offset);
    std::vector<Tensor> columns_lists = GetSplitTensor(&columns);
    std::vector<Tensor> sparse_dot_sdd_lists = GetSplitTensor(&sparse_dot_sdd);
    std::vector<Tensor> softmax_lists = GetSplitTensor(&softmax);
    std::vector<Tensor> dout_lists = GetSplitTensor(&dout);
    std::vector<Tensor> dquery_lists = GetSplitTensor(&dquery);
    std::vector<Tensor> dkey_lists = GetSplitTensor(&dkey);
    std::vector<Tensor> dvalue_lists = GetSplitTensor(&dvalue);

    const int iter_num = batch_size * num_heads;
    const auto& dev_ctx = ctx.cuda_device_context();
    for (int i = 0; i < iter_num; i++) {
      // dValue = transpose(result_softmax) * dOut
      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &softmax_lists[i], &dout_lists[i],
                               &dvalue_lists[i], M, N, true, false);

      // dSoftmax = dOut * transpose(Value)
      int nnz_num = columns.dims()[0];
      Tensor dsoftmax;
      dsoftmax.Resize({nnz_num});
      dsoftmax.mutable_data<T>(ctx.GetPlace());
      DotSdd<DeviceContext, T>(dev_ctx, &dout_lists[i], &value_lists[i],
                               &offset_lists[i], &columns_lists[i], &dsoftmax,
                               M, N, false, true);

      // dSparseDotSdd = dSoftmax * softmax'(SparseDotSdd)
      Tensor dsparse_dot_sdd;
      dsparse_dot_sdd.Resize({nnz_num});
      dsparse_dot_sdd.mutable_data<T>(ctx.GetPlace());
      SparseSoftmaxBackward<DeviceContext, T>(
          dev_ctx, &offset_lists[i], &columns_lists[i], &dsparse_dot_sdd,
          &dsoftmax, &softmax_lists[i], 1, M, N);

      // dQuery = dSparseDotSdd * Key
      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &dsparse_dot_sdd, &key_lists[i],
                               &dquery_lists[i], M, N, false, false);

      // dKey = transpose(dSparseDotSdd) * Query
      DotDsd<DeviceContext, T>(dev_ctx, &offset_lists[i], &columns_lists[i],
                               &dsparse_dot_sdd, &query_lists[i],
                               &dkey_lists[i], M, N, true, false);
    }
  }
};

}  // namespace operators
}  // namespace paddle
REGISTER_OP_CUDA_KERNEL(
    sparse_attention,
    ops::SparseAttentionCUDAKernel<plf::CUDADeviceContext, float>,
    ops::SparseAttentionCUDAKernel<plf::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    sparse_attention_grad,
    ops::SparseAttentionGradCUDAKernel<plf::CUDADeviceContext, float>,
    ops::SparseAttentionGradCUDAKernel<plf::CUDADeviceContext, double>);
