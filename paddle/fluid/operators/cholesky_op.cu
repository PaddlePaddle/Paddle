#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <algorithm>
#include <vector>
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/cholesky_op.h"
#include "paddle/fluid/platform/dynload/cusolver.h"

namespace paddle {
namespace operators {

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void MatrixBandPart(const int num_threads, const int m, const int n,
                               const int num_lower_diags,
                               const int num_upper_diags, const T* input_data,
                               T* output_data) {
  CUDA_1D_KERNEL_LOOP(index, num_threads) {
    const int col = index % n;
    const int row = (index / n) % m;
    const int band_start = (num_lower_diags < 0 ? 0 : row - num_lower_diags);
    const int band_end = (num_upper_diags < 0 ? n : row + num_upper_diags + 1);
    if (col < band_start || col >= band_end) {
      output_data[index] = T(0);
    } else {
      output_data[index] = input_data[index];
    }
  }
}

template <typename T>
class CholeskyGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto& dev_ctx =
        context.template device_context<paddle::platform::CUDADeviceContext>();

    const Tensor* x = context.Input<Tensor>("X");
    Tensor* out = context.Output<Tensor>("Out");

    bool upper = context.Attr<bool>("upper");
    auto& dims = x->dims();
    int batch_count = 1;
    for (int i = 0; i < dims.size() - 2; i++) {
      batch_count *= dims[i];
    }
    int m = dims[dims.size() - 1];
    int tensor_size = batch_count * m * m;

    const auto* x_data = x->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());

    // matrices are assumed to be stored in column-major order in cusolver
    hipblasFillMode_t uplo =
        upper ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
    // portf is inplace, thus copy the triangular part of the input matrices to
    // the output and set the other triangular part to 0 firstly
    int threads = std::min(1024, dev_ctx.GetMaxThreadsPerBlock());
    int blocks = (tensor_size + threads - 1) / threads;
    if (upper) {
      MatrixBandPart<<<blocks, threads, 0, dev_ctx.stream()>>>(
          tensor_size, m, m, /* num_lower_diags */ 0,
          /* num_upper_diags */ m, x_data, out_data);
    } else {
      MatrixBandPart<<<blocks, threads, 0, dev_ctx.stream()>>>(
          tensor_size, m, m, /* num_lower_diags */ m,
          /* num_upper_diags */ 0, x_data, out_data);
    }

    // TODO(guosheng): Add callback to check info
    auto info = memory::Alloc(dev_ctx, sizeof(int) * batch_count);
    auto* info_ptr = reinterpret_cast<int*>(info->ptr());

#if CUDA_VERSION >= 9020
    if (batch_count > 1) {
      std::vector<T*> output_ptrs;
      for (int i = 0; i < batch_count; i++) {
        output_ptrs.emplace_back(out_data + i * m * m);
      }
      thrust::device_vector<T*> dev_output_ptrs(output_ptrs.begin(),
                                                output_ptrs.end());
      PotrfBatched(dev_ctx, uplo, m,
                   thrust::raw_pointer_cast(dev_output_ptrs.data()), m,
                   info_ptr, batch_count);
      // TODO(guosheng): There seems to a bug in cusolver potrfBatched and need
      // to clear the upper triangle of the output. Remove this workaround once
      // the bug is fixed.
      if (!upper) {
        MatrixBandPart<<<blocks, threads, 0, dev_ctx.stream()>>>(
            tensor_size, m, m, /* num_lower_diags */ m,
            /* num_upper_diags */ 0, out_data, out_data);
      }
    } else {
#endif
      for (int i = 0; i < batch_count; i++) {
        Potrf(dev_ctx, uplo, m, out_data + i * m * m, m, info_ptr + i);
      }

#if CUDA_VERSION >= 9020
    }
#endif
  }

  void Potrf(const platform::CUDADeviceContext& dev_ctx, hipblasFillMode_t uplo,
             int n, T* A, int lda, int* info) const;

  void PotrfBatched(const platform::CUDADeviceContext& dev_ctx,
                    hipblasFillMode_t uplo, int n, T* Aarray[], int lda,
                    int* info_array, int batch_size) const;
};

#define FUNC_WITH_TYPES(m) m(float, S) m(double, D)

#define POTRF_INSTANCE(T, C)                                                   \
  template <>                                                                  \
  void CholeskyGPUKernel<T>::Potrf(const platform::CUDADeviceContext& dev_ctx, \
                                   hipblasFillMode_t uplo, int n, T* A,         \
                                   int lda, int* info) const {                 \
    auto handle = dev_ctx.cusolver_dn_handle();                                \
    int workspace_size = 0;                                                    \
    PADDLE_ENFORCE_CUDA_SUCCESS(                                               \
        platform::dynload::cusolverDn##C##potrf_bufferSize(                    \
            handle, uplo, n, A, lda, &workspace_size));                        \
    auto workspace = memory::Alloc(dev_ctx, workspace_size);                   \
    T* workspace_ptr = reinterpret_cast<T*>(workspace->ptr());                 \
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::cusolverDn##C##potrf(       \
        handle, uplo, n, A, lda, workspace_ptr, workspace_size, info));        \
  }

FUNC_WITH_TYPES(POTRF_INSTANCE);

#if CUDA_VERSION >= 9020
#define POTRF_BATCH_INSTANCE(T, C)                                          \
  template <>                                                               \
  void CholeskyGPUKernel<T>::PotrfBatched(                                  \
      const platform::CUDADeviceContext& dev_ctx, hipblasFillMode_t uplo,    \
      int n, T* Aarray[], int lda, int* info_array, int batch_size) const { \
    auto handle = dev_ctx.cusolver_dn_handle();                             \
    PADDLE_ENFORCE_CUDA_SUCCESS(                                            \
        platform::dynload::cusolverDn##C##potrfBatched(                     \
            handle, uplo, n, Aarray, lda, info_array, batch_size));         \
  }

FUNC_WITH_TYPES(POTRF_BATCH_INSTANCE);
#endif

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(cholesky, ops::CholeskyGPUKernel<float>,
                        ops::CholeskyGPUKernel<double>);
