/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifndef PADDLE_WITH_HIP
// HIP not support cusolver

#include <thrust/device_vector.h>
#include <algorithm>
#include <vector>
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/cholesky_op.h"
#include "paddle/fluid/platform/dynload/cusolver.h"

namespace paddle {
namespace operators {

template <typename T>
class CholeskyGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();

    const Tensor* x = context.Input<Tensor>("X");
    Tensor* out = context.Output<Tensor>("Out");

    bool upper = context.Attr<bool>("upper");
    auto& dims = x->dims();
    int batch_count = 1;
    for (int i = 0; i < dims.size() - 2; i++) {
      batch_count *= dims[i];
    }
    int m = dims[dims.size() - 1];
    int tensor_size = batch_count * m * m;

    const auto* x_data = x->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());

    // matrices are assumed to be stored in column-major order in cusolver
    hipblasFillMode_t uplo =
        upper ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
    // portf is inplace, thus copy the triangular part of the input matrices to
    // the output and set the other triangular part to 0 firstly
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx,
                                                              tensor_size);
    if (upper) {
      MatrixBandPartFunctor<T> matrix_band_part_functor(
          m, m, /* num_lower_diags */ 0, /* num_upper_diags */ m, x_data,
          out_data);
      for_range(matrix_band_part_functor);
    } else {
      MatrixBandPartFunctor<T> matrix_band_part_functor(
          m, m, /* num_lower_diags */ m, /* num_upper_diags */ 0, x_data,
          out_data);
      for_range(matrix_band_part_functor);
    }

    auto info = memory::Alloc(dev_ctx, sizeof(int) * batch_count);
    auto* info_ptr = reinterpret_cast<int*>(info->ptr());

#if CUDA_VERSION >= 9020 && !defined(_WIN32)
    if (batch_count > 1) {
      std::vector<T*> output_ptrs;
      for (int i = 0; i < batch_count; i++) {
        output_ptrs.emplace_back(out_data + i * m * m);
      }
      thrust::device_vector<T*> dev_output_ptrs(output_ptrs.begin(),
                                                output_ptrs.end());
      PotrfBatched(dev_ctx, uplo, m,
                   thrust::raw_pointer_cast(dev_output_ptrs.data()), m,
                   info_ptr, batch_count);
      // TODO(guosheng): There seems to a bug in cusolver potrfBatched and need
      // to clear the upper triangle of the output. Remove this workaround once
      // the bug is fixed.
      if (!upper) {
        MatrixBandPartFunctor<T> matrix_band_part_functor(
            m, m, /* num_lower_diags */ m, /* num_upper_diags */ 0, out_data,
            out_data);
        for_range(matrix_band_part_functor);
      }
    } else {
#endif
      for (int i = 0; i < batch_count; i++) {
        Potrf(dev_ctx, uplo, m, out_data + i * m * m, m, info_ptr + i);
      }

#if CUDA_VERSION >= 9020 && !defined(_WIN32)
    }
#endif
    // check the info
    std::vector<int> error_info;  // only for checking positive matrix
    error_info.resize(batch_count);

    memory::Copy(platform::CPUPlace(), error_info.data(),
                 BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                 info_ptr, sizeof(int) * batch_count, dev_ctx.stream());

    for (int i = 0; i < batch_count; ++i) {
      PADDLE_ENFORCE_EQ(error_info[i], 0,
                        platform::errors::PreconditionNotMet(
                            "For batch [%d]: U(%d, %d) is zero, singular U.", i,
                            error_info[i], error_info[i]));
    }
  }

  void Potrf(const platform::CUDADeviceContext& dev_ctx, hipblasFillMode_t uplo,
             int n, T* A, int lda, int* info) const;

  void PotrfBatched(const platform::CUDADeviceContext& dev_ctx,
                    hipblasFillMode_t uplo, int n, T* Aarray[], int lda,
                    int* info_array, int batch_size) const;
};

#define FUNC_WITH_TYPES(m) m(float, S) m(double, D)

#define POTRF_INSTANCE(T, C)                                                   \
  template <>                                                                  \
  void CholeskyGPUKernel<T>::Potrf(const platform::CUDADeviceContext& dev_ctx, \
                                   hipblasFillMode_t uplo, int n, T* A,         \
                                   int lda, int* info) const {                 \
    auto handle = dev_ctx.cusolver_dn_handle();                                \
    int workspace_size = 0;                                                    \
    PADDLE_ENFORCE_CUDA_SUCCESS(                                               \
        platform::dynload::cusolverDn##C##potrf_bufferSize(                    \
            handle, uplo, n, A, lda, &workspace_size));                        \
    auto workspace = memory::Alloc(dev_ctx, workspace_size);                   \
    T* workspace_ptr = reinterpret_cast<T*>(workspace->ptr());                 \
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::cusolverDn##C##potrf(       \
        handle, uplo, n, A, lda, workspace_ptr, workspace_size, info));        \
  }

FUNC_WITH_TYPES(POTRF_INSTANCE);

#if CUDA_VERSION >= 9020 && !defined(_WIN32)
#define POTRF_BATCH_INSTANCE(T, C)                                          \
  template <>                                                               \
  void CholeskyGPUKernel<T>::PotrfBatched(                                  \
      const platform::CUDADeviceContext& dev_ctx, hipblasFillMode_t uplo,    \
      int n, T* Aarray[], int lda, int* info_array, int batch_size) const { \
    auto handle = dev_ctx.cusolver_dn_handle();                             \
    PADDLE_ENFORCE_CUDA_SUCCESS(                                            \
        platform::dynload::cusolverDn##C##potrfBatched(                     \
            handle, uplo, n, Aarray, lda, info_array, batch_size));         \
  }

FUNC_WITH_TYPES(POTRF_BATCH_INSTANCE);
#endif

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(cholesky, ops::CholeskyGPUKernel<float>,
                        ops::CholeskyGPUKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    cholesky_grad,
    ops::CholeskyGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CholeskyGradKernel<paddle::platform::CUDADeviceContext, double>);

#endif  // not PADDLE_WITH_HIP
