/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/gather_nd_op.h"
#include "paddle/fluid/operators/scatter.cu.h"

namespace paddle {
namespace operators {

template <typename DeviceContext, typename T>
class GatherNdOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "This kernel only runs on GPU device.");
    auto *x = ctx.Input<Tensor>("X");
    auto *index = ctx.Input<Tensor>("Index");
    auto *output = ctx.Output<Tensor>("Out");

    output->mutable_data<T>(ctx.GetPlace());
    if (x->numel() == 0) return;
    const auto &index_type = index->type();
    bool index_type_match = index_type == framework::proto::VarType::INT32 ||
                            index_type == framework::proto::VarType::INT64;
    PADDLE_ENFORCE_EQ(
        index_type_match, true,
        "Index holds the wrong type, it holds %s, but desires to be %s or %s",
        paddle::framework::DataTypeToString(index_type),
        paddle::framework::DataTypeToString(framework::proto::VarType::INT32),
        paddle::framework::DataTypeToString(framework::proto::VarType::INT64));
    if (index_type == framework::proto::VarType::INT32) {
      GPUGatherNd<DeviceContext, T, int>(ctx, *x, *index, output);
    } else if (index_type == framework::proto::VarType::INT64) {
      GPUGatherNd<DeviceContext, T, int64_t>(ctx, *x, *index, output);
    }
  }
};

template <typename DeviceContext, typename T>
class GatherNdGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "This kernel only runs on GPU device.");
    auto *index = ctx.Input<Tensor>("Index");
    auto *dX = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *dO = ctx.Input<Tensor>(framework::GradVarName("Out"));

    dX->mutable_data<T>(ctx.GetPlace());
    auto dxt = framework::EigenVector<T>::Flatten(*dX);
    auto &place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();
    dxt.device(place) = dxt.constant(static_cast<T>(0));
    if (dO->numel() == 0) return;

    const auto &index_type = index->type();
    bool index_type_match = index_type == framework::proto::VarType::INT32 ||
                            index_type == framework::proto::VarType::INT64;

    PADDLE_ENFORCE_EQ(
        index_type_match, true,
        "Index holds the wrong type, it holds %s, but desires to be %s or %s",
        paddle::framework::DataTypeToString(index_type),
        paddle::framework::DataTypeToString(framework::proto::VarType::INT32),
        paddle::framework::DataTypeToString(framework::proto::VarType::INT64));

    if (index_type == framework::proto::VarType::INT32) {
      GPUScatterNdAdd<DeviceContext, T, int>(ctx, *dO, *index, dX);
    } else if (index_type == framework::proto::VarType::INT64) {
      GPUScatterNdAdd<DeviceContext, T, int64_t>(ctx, *dO, *index, dX);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(gather_nd, ops::GatherNdOpCUDAKernel<CUDA, float>,
                        ops::GatherNdOpCUDAKernel<CUDA, double>,
                        ops::GatherNdOpCUDAKernel<CUDA, int64_t>,
                        ops::GatherNdOpCUDAKernel<CUDA, int>,
                        ops::GatherNdOpCUDAKernel<CUDA, plat::float16>);

REGISTER_OP_CUDA_KERNEL(gather_nd_grad,
                        ops::GatherNdGradOpCUDAKernel<CUDA, float>,
                        ops::GatherNdGradOpCUDAKernel<CUDA, double>,
                        ops::GatherNdGradOpCUDAKernel<CUDA, int64_t>,
                        ops::GatherNdGradOpCUDAKernel<CUDA, int>,
                        ops::GatherNdGradOpCUDAKernel<CUDA, plat::float16>);
