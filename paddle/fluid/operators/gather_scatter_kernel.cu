#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/gather_scatter_kernel.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

class TensorAssign {
 public:
  template <typename tensor_t>
  constexpr void operator()(tensor_t* self_data, tensor_t* src_data) const {
    *self_data = *src_data;
  }
};
static TensorAssign tensor_assign;

class ReduceAdd {
 public:
  template <
      typename tensor_t,
      std::enable_if_t<!std::is_same<tensor_t, uint8_t>::value>* = nullptr>
  __device__ void operator()(tensor_t* self_data, tensor_t* src_data) const {
    platform::CudaAtomicAdd(self_data, *src_data);
  }
  template <typename tensor_t,
            std::enable_if_t<std::is_same<tensor_t, uint8_t>::value>* = nullptr>
  __device__ void operator()(tensor_t* self_data, tensor_t* src_data) const {
    *self_data += *src_data;
  }
};
static ReduceAdd reduce_add;

class ReduceMul {
 public:
  template <typename tensor_t>
  __device__ void operator()(tensor_t* self_data, tensor_t* src_data) const {
    *self_data *= *src_data;
    // TODO(huangxu96) platform::CudaAtomicMul(*self_data, *src_data);
  }
};
static ReduceMul reduce_mul;

template <typename tensor_t, typename index_t, typename func_t,
          bool is_scatter_like = true>
__global__ void GatherScatterGPUKernel(
    tensor_t* self_data, int dim, const index_t* index_data, tensor_t* src_data,
    int64_t inner_dim_size, int select_dim_size, int replaced_select_dim_size,
    int64_t outer_dim_size, int64_t numel, const func_t& reduce_op) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numel) return;
  int64_t i, j, k;  // The i, j, k here is the index of the 3 layers loop
                    // squeezed from the N layers loop.
  /* tid = i * select_dim_size * outer_dim_size + j * outer_dim_size + k */
  i = tid / (select_dim_size * outer_dim_size);
  int64_t remind = tid % (select_dim_size * outer_dim_size);
  j = remind / outer_dim_size;
  k = remind % outer_dim_size;
  index_t index = index_data[tid];
  /*
    gather computation formula:

    self[i][j][k] = src[index[i][j][k]][j][k]  # if dim == 0
    self[i][j][k] = src[i][index[i][j][k]][k]  # if dim == 1
    self[i][j][k] = src[i][j][index[i][j][k]]  # if dim == 2

    scatter computation formula:

    self[index[i][j][k]][j][k] = src[i][j][k]  # if dim == 0
    self[i][index[i][j][k]][k] = src[i][j][k]  # if dim == 1
    self[i][j][index[i][j][k]] = src[i][j][k]  # if dim == 2

  */
  // index matrix has different shape with self matrix or src matrix.
  int64_t replace_index = k + index * outer_dim_size +
                          i * outer_dim_size * replaced_select_dim_size;
  int64_t self_idx = is_scatter_like ? replace_index : tid;
  int64_t src_idx = is_scatter_like ? tid : replace_index;
  reduce_op((tensor_t*)(self_data + self_idx), (tensor_t*)(src_data + src_idx));
}

template <typename tensor_t, typename index_t = int64_t,
          bool is_scatter_like = true>
struct gpu_gather_scatter_functor {
  template <typename func_t>
  void operator()(Tensor self, int dim, const Tensor& index, Tensor src,
                  const std::string& method_name, const func_t& reduce_op,
                  const platform::DeviceContext& ctx) {
    if (index.numel() == 0) {
      return;
    }
    auto* self_data = self.data<tensor_t>();
    auto* index_data = index.data<index_t>();
    auto* src_data = src.data<tensor_t>();
    int64_t self_size = self.numel();
    int64_t index_size = index.numel();
    int64_t src_size = src.numel();
    auto self_dims = self.dims();
    auto index_dims = index.dims();
    auto src_dims = src.dims();
    if (self_size == 0 || src_size == 0 || index_size == 0) return;
    int select_dim_size = index_dims[dim];
    // index matrix has different shape with self matrix or src matrix.
    int replaced_select_dim_size =
        is_scatter_like ? self_dims[dim] : src_dims[dim];
    int64_t inner_dim_size = 1;
    int64_t outer_dim_size = 1;
    for (int64_t i = 0; i < index_dims.size(); ++i) {
      inner_dim_size *= index_dims[i];
    }

    for (int i = dim + 1; i < index_dims.size(); i++) {
      outer_dim_size *= index_dims[i];
    }

    int64_t slice_size = 1;
    for (int i = 1; i < src_dims.size(); ++i) slice_size *= src_dims[i];

    int block = 512;
    int64_t n = slice_size * index_size;
    int64_t grid = (n + block - 1) / block;
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    GatherScatterGPUKernel<tensor_t, index_t, func_t,
                           is_scatter_like><<<grid, block, 0, stream>>>(
        self_data, dim, index_data, src_data, inner_dim_size, select_dim_size,
        replaced_select_dim_size, outer_dim_size, index_size, reduce_op);
  }
};  // struct gpu_gather_scatter_functor

template <typename tensor_t, typename index_t>
void gpu_gather_kernel(Tensor self, int dim, const Tensor& index, Tensor result,
                       const platform::DeviceContext& ctx) {
  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/false>()(
      result, dim, index, self, "gather_out_gpu", tensor_assign, ctx);
  return;
}

template <typename tensor_t, typename index_t>
void gpu_scatter_assign_kernel(Tensor self, int dim, const Tensor& index,
                               Tensor src, const platform::DeviceContext& ctx) {
  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/true>()(
      self, dim, index, src, "scatter_assign_gpu", tensor_assign, ctx);
}

template <typename tensor_t, typename index_t>
void gpu_scatter_add_kernel(Tensor self, int dim, const Tensor& index,
                            Tensor src, const platform::DeviceContext& ctx) {
  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/true>()(
      self, dim, index, src, "scatter_add_gpu", reduce_add, ctx);
}

template <typename tensor_t, typename index_t>
void gpu_scatter_mul_kernel(Tensor self, int dim, const Tensor& index,
                            Tensor src, const platform::DeviceContext& ctx) {
  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/true>()(
      self, dim, index, src, "scatter_mul_gpu", reduce_mul, ctx);
}

template <typename tensor_t, typename index_t>
__global__ void ScatterInputGradGPUKernel(
    tensor_t* grad_data, int dim, const index_t* index_data,
    int64_t inner_dim_size, int select_dim_size, int grad_select_dim_size,
    int64_t outer_dim_size, int64_t numel) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numel) return;
  int64_t i, j, k;
  i = tid / (select_dim_size * outer_dim_size);
  int64_t remind = tid % (select_dim_size * outer_dim_size);
  j = remind / outer_dim_size;
  k = remind % outer_dim_size;
  index_t index = index_data[tid];
  int64_t replace_index =
      k + index * outer_dim_size + i * outer_dim_size * grad_select_dim_size;
  grad_data[replace_index] = 0;
}
template <typename tensor_t, typename index_t>
void gpu_scatter_input_grad_kernel(Tensor self, int dim, const Tensor& index,
                                   Tensor grad,
                                   const platform::DeviceContext& ctx) {
  auto* index_data = index.data<index_t>();
  auto* grad_data = grad.data<tensor_t>();

  auto index_dims = index.dims();
  auto grad_dims = grad.dims();
  int64_t index_size = index.numel();

  int64_t inner_dim_size = 1;
  int64_t outer_dim_size = 1;
  int select_dim_size = index_dims[dim];
  int grad_select_dim_size = grad_dims[dim];
  for (int64_t i = 0; i < dim; ++i) {
    inner_dim_size *= index_dims[i];
  }

  for (int i = dim + 1; i < index_dims.size(); i++) {
    outer_dim_size *= index_dims[i];
  }

  int64_t slice_size = 1;
  for (int i = 1; i < grad_dims.size(); ++i) slice_size *= grad_dims[i];

  int block = 512;
  int64_t n = slice_size * index_size;
  int64_t grid = (n + block - 1) / block;
  auto stream =
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();

  ScatterInputGradGPUKernel<tensor_t, index_t><<<grid, block, 0, stream>>>(
      grad_data, dim, index_data, inner_dim_size, select_dim_size,
      grad_select_dim_size, outer_dim_size, index_size);
}
Instantiate_Template_Function(gpu_gather_kernel)
    Instantiate_Template_Function(gpu_scatter_assign_kernel)
        Instantiate_Template_Function(gpu_scatter_add_kernel)
            Instantiate_Template_Function(gpu_scatter_mul_kernel)
                Instantiate_Template_Function(gpu_scatter_input_grad_kernel)

}  // namespace operators
}  // namespace paddle
