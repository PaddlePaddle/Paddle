#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/gather_scatter_kernel.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

class TensorAssign {
 public:
  template <typename tensor_t>
  constexpr void operator()(tensor_t* self_data, tensor_t* src_data) const {
    *self_data = *src_data;
  }
};
static TensorAssign tensor_assign;

class ReduceMul {
 public:
  template <typename tensor_t>
  __device__ void operator()(tensor_t* self_data, tensor_t* src_data) const {
    *self_data *= *src_data;
    // TODO(huangxu96) platform::CudaAtomicMul(*self_data, *src_data);
  }
};
static ReduceMul reduce_mul;

class ReduceAdd {
 public:
  template <
      typename tensor_t,
      std::enable_if_t<!std::is_same<tensor_t, uint8_t>::value>* = nullptr>
  __device__ void operator()(tensor_t* self_data, tensor_t* src_data) const {
    platform::CudaAtomicAdd(self_data, *src_data);
  }
  template <typename tensor_t,
            std::enable_if_t<std::is_same<tensor_t, uint8_t>::value>* = nullptr>
  __device__ void operator()(tensor_t* self_data, tensor_t* src_data) const {
    *self_data += *src_data;
  }
};
static ReduceAdd reduce_add;

template <typename tensor_t, typename index_t, typename func_t,
          bool is_scatter_like = true>
__global__ void GatherScatterGPUKernel(
    tensor_t* self_data, int dim, const index_t* index_data, tensor_t* src_data,
    int64_t inner_dim_size, int select_dim_size, int replaced_select_dim_size,
    int64_t outer_dim_size, int64_t numel, const func_t& reduce_op) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numel) return;
  int64_t i, j, k;  // The i, j, k here is the index of the 3 layers loop
                    // squeezed from the N layers loop.
  /* tid = i * select_dim_size * outer_dim_size + j * outer_dim_size + k */
  i = tid / (select_dim_size * outer_dim_size);
  int64_t remind = tid % (select_dim_size * outer_dim_size);
  j = remind / outer_dim_size;
  k = remind % outer_dim_size;
  index_t index = index_data[tid];
  /*
    gather computation formula:

    self[i][j][k] = src[index[i][j][k]][j][k]  # if dim == 0
    self[i][j][k] = src[i][index[i][j][k]][k]  # if dim == 1
    self[i][j][k] = src[i][j][index[i][j][k]]  # if dim == 2

    scatter computation formula:

    self[index[i][j][k]][j][k] = src[i][j][k]  # if dim == 0
    self[i][index[i][j][k]][k] = src[i][j][k]  # if dim == 1
    self[i][j][index[i][j][k]] = src[i][j][k]  # if dim == 2

  */
  // index matrix has different shape with self matrix or src matrix.
  int64_t replace_index = k + index * outer_dim_size +
                          i * outer_dim_size * replaced_select_dim_size;
  int64_t self_idx = is_scatter_like ? replace_index : tid;
  int64_t src_idx = is_scatter_like ? tid : replace_index;
  reduce_op((tensor_t*)(self_data + self_idx), (tensor_t*)(src_data + src_idx));
}

template <typename tensor_t, typename index_t = int64_t,
          bool is_scatter_like = true>
struct gpu_gather_scatter_functor {
  template <typename func_t>
  void operator()(Tensor self, int dim, const Tensor& index, Tensor src,
                  const std::string& method_name, const func_t& reduce_op,
                  const platform::DeviceContext& ctx) {
    if (index.numel() == 0) {
      return;
    }
    auto* self_data = self.data<tensor_t>();
    auto* index_data = index.data<index_t>();
    auto* src_data = src.data<tensor_t>();
    int64_t self_size = self.numel();
    int64_t index_size = index.numel();
    int64_t src_size = src.numel();
    auto self_dims = self.dims();
    auto index_dims = index.dims();
    auto src_dims = src.dims();
    if (self_size == 0 || src_size == 0 || index_size == 0) return;
    int select_dim_size = index_dims[dim];
    // index matrix has different shape with self matrix or src matrix.
    int replaced_select_dim_size =
        is_scatter_like ? self_dims[dim] : src_dims[dim];
    int64_t inner_dim_size = 1;
    int64_t outer_dim_size = 1;
    for (int64_t i = 0; i < index_dims.size(); ++i) {
      inner_dim_size *= index_dims[i];
    }

    for (int i = dim + 1; i < index_dims.size(); i++) {
      outer_dim_size *= index_dims[i];
    }

    int64_t slice_size = 1;
    for (int i = 1; i < src_dims.size(); ++i) slice_size *= src_dims[i];

    int block = 512;
    int64_t n = slice_size * index_size;
    int64_t grid = (n + block - 1) / block;
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    GatherScatterGPUKernel<tensor_t, index_t, func_t,
                           is_scatter_like><<<grid, block, 0, stream>>>(
        self_data, dim, index_data, src_data, inner_dim_size, select_dim_size,
        replaced_select_dim_size, outer_dim_size, index_size, reduce_op);
  }
};  // struct gpu_gather_scatter_functor

template <typename tensor_t, typename index_t>
void gpu_gather_kernel(Tensor self, int dim, const Tensor& index, Tensor result,
                       const platform::DeviceContext& ctx) {
  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/false>()(
      result, dim, index, self, "gather_out_gpu", tensor_assign, ctx);
  return;
}

template <typename tensor_t, typename index_t>
void gpu_scatter_assign_kernel(Tensor self, int dim, const Tensor& index,
                               Tensor src, const platform::DeviceContext& ctx) {
  VLOG(3) << "start scatter assign kernel";

  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/true>()(
      self, dim, index, src, "scatter_assign_gpu", tensor_assign, ctx);
  VLOG(3) << "<<<< Done gpu_scatter_add_kernel <<<<<";
}

template <typename tensor_t, typename index_t>
void gpu_scatter_add_kernel(Tensor self, int dim, const Tensor& index,
                            Tensor src, const platform::DeviceContext& ctx) {
  VLOG(3) << "start scatter add kernel";

  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/true>()(
      self, dim, index, src, "scatter_add_gpu", reduce_add, ctx);
  VLOG(3) << "<<<< Done gpu_scatter_add_kernel <<<<<";
}

template <typename tensor_t, typename index_t>
void gpu_scatter_mul_kernel(Tensor self, int dim, const Tensor& index,
                            Tensor src, const platform::DeviceContext& ctx) {
  VLOG(3) << "start scatter mul kernel";

  gpu_gather_scatter_functor<tensor_t, index_t,
                             /*is_scatter_like=*/true>()(
      self, dim, index, src, "scatter_mul_gpu", reduce_mul, ctx);
  VLOG(3) << "<<<< Done gpu_scatter_mul_kernel <<<<<";
}

namespace plat = paddle::platform;
Instantiate_Template_Funtion(gpu_gather_kernel)
    Instantiate_Template_Funtion(gpu_scatter_assign_kernel)
        Instantiate_Template_Funtion(gpu_scatter_add_kernel)
            Instantiate_Template_Funtion(gpu_scatter_mul_kernel)

}  // namespace operators
}  // namespace paddle
