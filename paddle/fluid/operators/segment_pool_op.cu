/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/segment_pool_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_param_config.h"

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    segment_pool,
    ops::SegmentPoolKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SegmentPoolKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    segment_pool_grad,
    ops::SegmentPoolGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SegmentPoolGradKernel<paddle::platform::CUDADeviceContext, double>);
