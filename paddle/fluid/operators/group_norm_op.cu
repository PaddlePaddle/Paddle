#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/operators/group_norm_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

using DataLayout = framework::DataLayout;
enum GroupNormKernelFlags { kHasScale = 1, kHasBias = 2 };
#define ALIGN_BYTES 16

#define CHECK_CASE(i, flags, kernel_name, ...)                              \
  if (i == flags) {                                                         \
    kernel_name<T, i><<<grid, threads, 0, dev_ctx.stream()>>>(__VA_ARGS__); \
  }

// 0 for no scale, no bias
// 1 for has scale, no bias
// 2 for no scale, has bias
// 3 for has scale, has bias
#define UNROLL_ALL_CASES(flags, kernel_name, ...) \
  CHECK_CASE(0, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(1, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(2, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(3, flags, kernel_name, __VA_ARGS__)

template <typename T>
__device__ __inline__ void CudaAtomicAddWithWarp(T* sum, T value) {
  typedef hipcub::WarpReduce<T> WarpReduce;
  typename WarpReduce::TempStorage temp_storage;
  value = WarpReduce(temp_storage).Sum(value);
  if (hipcub::LaneId() == 0) platform::CudaAtomicAdd(sum, value);
}

template <typename T>
__global__ void GroupNormForwardGetMeanAndVar(const T* x,
                                              int N,
                                              int C,
                                              int W,
                                              int imsize,
                                              int groups,
                                              int group_size,
                                              T* mean,
                                              T* var) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_mean = 0, x_var = 0;
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    int hid = imid / W;
    int wid = imid % W;
    val = x[(bid * H + hid) * W * C + wid * C + ccid];

    x_mean += val;
    x_var += val * val;
  }
  x_mean /= number * imsize;
  x_var /= number * imsize;
  CudaAtomicAddWithWarp(&mean[bid * groups + gid], x_mean);
  CudaAtomicAddWithWarp(&var[bid * groups + gid], x_var);
}

template <typename T, typename AccT, int VecSize, int Num>
__device__ __forceinline__ void ThreadReduce(phi::Array<const T*, Num> arrs,
                                             int size,
                                             const int offset,
                                             AccT* out_mean,
                                             AccT* out_var) {
  const T* x = arrs[0];
  const T* y;
  if (Num == 2) {
    y = arrs[1];
  }
  using VecT = kps::details::VectorType<T, VecSize>;
  int tid = threadIdx.x;
  if (offset > 0) {
    x -= offset;
    if (Num == 2) {
      y -= offset;
    }
    size += offset;
    if (tid >= offset) {
      if (Num == 1) {
        *out_mean += x[tid];
        *out_var += x[tid] * x[tid];
      } else if (Num == 2) {
        *out_mean += y[tid];
        *out_var += y[tid] * x[tid];
      }
    }
    size -= blockDim.x;
    x += blockDim.x;
    if (Num == 2) {
      y += blockDim.x;
    }
  }
  int remain = size % (VecSize * blockDim.x);

  T ins_x[VecSize];
  T ins_y[VecSize];
  VecT* ins_vec_x = reinterpret_cast<VecT*>(&ins_x);
  VecT* ins_vec_y = reinterpret_cast<VecT*>(&ins_y);

  // vector part
  for (; VecSize * tid < (size - remain); tid += blockDim.x) {
    *ins_vec_x = reinterpret_cast<const VecT*>(x)[tid];
    if (Num == 2) {
      *ins_vec_y = reinterpret_cast<const VecT*>(y)[tid];
    }

#pragma unroll
    for (int i = 0; i < VecSize; ++i) {
      if (Num == 1) {
        *out_mean += ins_x[i];
        *out_var += ins_x[i] * ins_x[i];
      } else if (Num == 2) {
        *out_mean += ins_y[i];
        *out_var += ins_y[i] * ins_x[i];
      }
    }
  }

  // scalar part
  tid = size - remain + threadIdx.x;
  for (; tid < size; tid += blockDim.x) {
    if (Num == 1) {
      *out_mean += x[tid];
      *out_var += x[tid] * x[tid];
    } else if (Num == 2) {
      *out_mean += y[tid];
      *out_var += y[tid] * x[tid];
    }
  }
}

template <typename T>
__device__ __forceinline__ void ReduceMeanAndVar(
    T* mean, T* var, T x_mean, T x_var, int size) {
  const int nc = blockIdx.x;
  x_mean = kps::details::BlockXReduce<T, kps::AddFunctor<T>>(
      x_mean, kps::AddFunctor<T>());
  x_var = kps::details::BlockXReduce<T, kps::AddFunctor<T>>(
      x_var, kps::AddFunctor<T>());
  __syncthreads();
  if (threadIdx.x == 0) {
    mean[nc] = static_cast<T>(x_mean / size);
    var[nc] = static_cast<T>(x_var / size);
  }
}

template <typename T>
__global__ void ScalarGetMeanAndVarNCHW(const T* x, T* mean, T* var, int size) {
  int i = blockIdx.x;
  T x_mean = 0, x_var = 0;
  for (int j = threadIdx.x; j < size; j += blockDim.x) {
    T val;
    val = x[i * size + j];
    x_mean += val;
    x_var += val * val;
  }
  ReduceMeanAndVar<T>(mean, var, x_mean, x_var, size);
}

template <typename T, typename AccT, int VecSize>
__global__ void VectorizedGetMeanAndVarNCHW(const T* x,
                                            T* mean,
                                            T* var,
                                            int size) {
  int i = blockIdx.x;
  AccT x_mean = static_cast<AccT>(0);
  AccT x_var = static_cast<AccT>(0);
  x += i * size;
  const int input_offset = ((uint64_t)x) % ALIGN_BYTES / sizeof(T);
  phi::Array<const T*, 1> ins;
  ins[0] = x;
  ThreadReduce<T, AccT, VecSize, 1>(ins, size, input_offset, &x_mean, &x_var);
  ReduceMeanAndVar<AccT>(mean, var, x_mean, x_var, size);
}

template <typename T, int flags>
__global__ void GroupNormForward(const T* x,
                                 const T* mean,
                                 const T* var,
                                 const T* scale,
                                 const T* bias,
                                 int N,
                                 int C,
                                 int W,
                                 int imsize,
                                 int groups,
                                 int group_size,
                                 T epsilon,
                                 T* y,
                                 T* real_var,
                                 const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  auto ng = bid * groups + gid;
  T x_mean = mean[ng];
  T x_var = var[ng];
  x_var = x_var - x_mean * x_mean;
  T var_inv = rsqrt(x_var + epsilon);
  if (cid == 0 && threadIdx.x == 0) {
    real_var[ng] = x_var;
  }
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    int hid, wid;
    int index = (bid * C + ccid) * imsize + imid;
    if (data_layout == DataLayout::kNCHW) {
      val = x[index];
    } else {
      hid = imid / W;
      wid = imid % W;
      val = x[(bid * H + hid) * W * C + wid * C + ccid];
    }
    val = (val - x_mean) * var_inv;
    if (flags & kHasScale) {
      val *= scale[ccid];
    }
    if (flags & kHasBias) {
      val += bias[ccid];
    }
    if (data_layout == DataLayout::kNCHW) {
      y[index] = val;
    } else {
      y[(bid * H + hid) * W * C + wid * C + ccid] = val;
    }
  }
}

template <typename T>
class GroupNormKernel<phi::GPUContext, T> : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const float epsilon = ctx.Attr<float>("epsilon");
    auto* scale = ctx.Input<Tensor>("Scale");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* x = ctx.Input<Tensor>("X");

    auto* y = ctx.Output<Tensor>("Y");
    auto* mean = ctx.Output<Tensor>("Mean");
    auto* var = ctx.Output<Tensor>("Variance");
    const auto groups = ctx.Attr<int>("groups");

    const auto x_dims = x->dims();
    const int C =
        (data_layout == DataLayout::kNCHW ? x_dims[1]
                                          : x_dims[x_dims.size() - 1]);
    const int group_size = C / groups;

    const int W =
        (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                          : x_dims[x_dims.size() - 2]);

    y->mutable_data<T>(ctx.GetPlace());
    mean->mutable_data<T>(ctx.GetPlace());
    var->mutable_data<T>(ctx.GetPlace());
    phi::funcs::SetConstant<phi::GPUContext, T> set_zero;
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();
    Tensor temp_var;
    temp_var.mutable_data<T>(var->dims(), ctx.GetPlace());
    auto* x_data = x->data<T>();
    auto* y_data = y->data<T>();
    auto* mean_data = mean->data<T>();
    auto* var_data = var->data<T>();
    auto* temp_var_data = temp_var.data<T>();

    const T* scale_data = nullptr;
    if (scale) scale_data = scale->data<T>();
    const T* bias_data = nullptr;
    if (bias) bias_data = bias->data<T>();

    int imsize = 1;
    if (data_layout == DataLayout::kNCHW) {
      for (int i = 2; i < x_dims.size(); ++i) {
        imsize *= x_dims[i];
      }
    } else {
      for (int i = 1; i < x_dims.size() - 1; ++i) {
        imsize *= x_dims[i];
      }
    }

#ifdef __HIPCC__
    int block_size = std::max(std::min(256, imsize), 64);
#else
    int block_size = std::min(1024, imsize);
#endif

    dim3 grid(group_size, groups, x_dims[0]);
    dim3 threads(block_size, 1, 1);
    if (data_layout == DataLayout::kNCHW) {
      using AccT = typename details::MPTypeTrait<T>::Type;
      constexpr int vec_size = sizeof(float4) / sizeof(T);
      int size = group_size * imsize;
      const int max_num_threads = 1024;
      int max_block_size = std::min(size / vec_size, max_num_threads);
      int block_size_nchw = 1;
      while (block_size_nchw < max_block_size) {
        block_size_nchw *= 2;
      }
      block_size_nchw = std::max(block_size_nchw, kps::details::kWarpSize);
      dim3 grids(x_dims[0] * groups);
      dim3 blocks(block_size_nchw);
      if (size < vec_size * block_size_nchw) {
        ScalarGetMeanAndVarNCHW<T><<<grids, blocks, 0, dev_ctx.stream()>>>(
            x_data, mean_data, temp_var_data, size);
      } else {
        VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
            <<<grids, blocks, 0, dev_ctx.stream()>>>(
                x_data, mean_data, temp_var_data, size);
      }
    } else {
      set_zero(dev_ctx, mean, static_cast<T>(0));
      set_zero(dev_ctx, &temp_var, static_cast<T>(0));
      GroupNormForwardGetMeanAndVar<T>
          <<<grid, threads, 0, dev_ctx.stream()>>>(x_data,
                                                   x_dims[0],
                                                   C,
                                                   W,
                                                   imsize,
                                                   groups,
                                                   group_size,
                                                   mean_data,
                                                   temp_var_data);
    }
    int flags =
        (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
    UNROLL_ALL_CASES(flags,
                     GroupNormForward,
                     x_data,
                     mean_data,
                     temp_var_data,
                     scale_data,
                     bias_data,
                     x_dims[0],
                     C,
                     W,
                     imsize,
                     groups,
                     group_size,
                     epsilon,
                     y_data,
                     var_data,
                     data_layout);
  }
};

template <typename T, int flags>
__global__ void GroupNormBackwardGetMeanAndVar(const T* x,
                                               const T* scale,
                                               const T* bias,
                                               const T* d_y,
                                               int N,
                                               int C,
                                               int W,
                                               int imsize,
                                               int groups,
                                               int group_size,
                                               T epsilon,
                                               T* d_mean,
                                               T* d_var,
                                               T* d_scale,
                                               T* d_bias) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_scale = (flags & kHasScale) ? scale[ccid] : 1;
  T x_bias = (flags & kHasBias) ? bias[ccid] : 0;
  T x_scale_inv = 0;
  if (x_scale != 0) x_scale_inv = 1.0 / x_scale;
  T d_mean_data = 0, d_var_data = 0, d_scale_data = 0, d_bias_data = 0;

  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val, dval;

    int hid = imid / W;
    int wid = imid % W;
    val = x[(bid * H + hid) * W * C + wid * C + ccid] - x_bias;
    dval = d_y[(bid * H + hid) * W * C + wid * C + ccid];

    d_var_data += val * dval;
    d_mean_data += dval * x_scale;

    val = val * x_scale_inv;
    d_bias_data += dval;
    d_scale_data += val * dval;
  }
  CudaAtomicAddWithWarp(&(d_mean[bid * groups + gid]), d_mean_data);
  CudaAtomicAddWithWarp(&(d_var[bid * groups + gid]), d_var_data);

  if (flags & kHasScale) {
#if CUDA_VERSION >= 11070
    platform::CudaAtomicAdd(&(d_scale[ccid]), d_scale_data);
#else
    CudaAtomicAddWithWarp(&(d_scale[ccid]), d_scale_data);
#endif
  }
  if (flags & kHasBias) {
#if CUDA_VERSION >= 11070
    platform::CudaAtomicAdd(&(d_bias[ccid]), d_bias_data);
#else
    CudaAtomicAddWithWarp(&(d_bias[ccid]), d_bias_data);
#endif
  }
}

template <typename T, int flags>
__global__ void GroupNormBackward(const T* x,
                                  const T* d_y,
                                  const T* scale,
                                  const T* bias,
                                  const T* var,
                                  const T* d_mean,
                                  const T* d_var,
                                  int N,
                                  int C,
                                  int W,
                                  int imsize,
                                  int groups,
                                  int group_size,
                                  T epsilon,
                                  T* d_x) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_var = var[bid * groups + gid];
  T d_x_mean = d_mean[bid * groups + gid];
  T d_x_var = d_var[bid * groups + gid];

  T x_var_inv = 1.0 / sqrt(x_var + epsilon);
  T number_inv = 1.0 / (number * imsize);

  T x_scale = (flags & kHasScale) ? scale[ccid] : 1;
  T x_bias = (flags & kHasBias) ? bias[ccid] : 0;
  T x_scale_inv = 0;
  if (x_scale != 0) x_scale_inv = 1.0 / x_scale;

  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    int hid = imid / W;
    int wid = imid % W;
    T tmp = x[(bid * H + hid) * W * C + wid * C + ccid];
    T v_y = (tmp - x_bias) * x_scale_inv;
    T dly = d_y[(bid * H + hid) * W * C + wid * C + ccid];
    d_x[(bid * H + hid) * W * C + wid * C + ccid] =
        x_var_inv *
        (dly * x_scale - number_inv * d_x_var * v_y - number_inv * d_x_mean);
  }
}

template <typename T>
__global__ void ScalarGetDsDbCUDAKernel(
    int imsize, const T* x, const T* dy, T* ds, T* db) {
  const int nc = blockIdx.x;
  T ds_sum = 0;
  T db_sum = 0;
  for (int i = threadIdx.x; i < imsize; i += blockDim.x) {
    const int index = nc * imsize + i;
    ds_sum += dy[index] * x[index];
    db_sum += dy[index];
  }
  ReduceMeanAndVar<T>(db, ds, db_sum, ds_sum, 1);
}

template <typename T>
__global__ void GetScaleBiasGradientCUDAKernel(int N,
                                               int C,
                                               int group,
                                               T epsilon,
                                               const T* mean,
                                               const T* var,
                                               const T* ds,
                                               const T* db,
                                               T* d_scale,
                                               T* d_bias) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < C) {
    const int G = group;
    const int D = C / G;
    T sum1 = 0;
    T sum2 = 0;
    for (int n = 0; n < N; ++n) {
      const int nc = n * C + c;
      const int ng = n * G + c / D;
      sum1 += (d_scale == nullptr)
                  ? T(0)
                  : ((ds[nc] - db[nc] * static_cast<T>(mean[ng])) *
                     static_cast<T>(rsqrt(var[ng] + epsilon)));
      sum2 += (d_bias == nullptr) ? T(0) : db[nc];
    }
    if (d_scale != nullptr) {
      d_scale[c] = sum1;
    }
    if (d_bias != nullptr) {
      d_bias[c] = sum2;
    }
  }
}

template <typename T, int BlockDim>
__global__ void GetBackwardParamsCUDAKernel(int imsize,
                                            int groups,
                                            int group_size,
                                            T epsilon,
                                            const T* mean,
                                            const T* var,
                                            const T* scale,
                                            const T* ds,
                                            const T* db,
                                            T* p1,
                                            T* p2,
                                            T* p3) {
  const int n = blockIdx.x;
  const int g = blockIdx.y;
  const int ng = n * groups + g;
  T sum1 = 0;
  T sum2 = 0;
  T var_inv = rsqrt(var[ng] + epsilon);
  for (int64_t i = threadIdx.x; i < group_size; i += blockDim.x) {
    const int64_t index = ng * group_size + i;
    const int64_t c = g * group_size + i;
    const T scale_v = scale == nullptr ? T(1) : static_cast<T>(scale[c]);
    sum1 += ds[index] * scale_v;
    sum2 += db[index] * scale_v;
    const T scale_c = scale == nullptr ? T(0) : static_cast<T>(scale[c]);
    p1[index] = scale_c * var_inv;
  }

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ds_storage;
  __shared__ typename BlockReduce::TempStorage db_storage;
  sum1 = BlockReduce(ds_storage).Reduce(sum1, hipcub::Sum());
  sum2 = BlockReduce(db_storage).Reduce(sum2, hipcub::Sum());

  if (threadIdx.x == 0) {
    const T s = T(1) / static_cast<T>(group_size * imsize);
    const T x = (sum2 * static_cast<T>(mean[ng]) - sum1) *
                static_cast<T>(var_inv) * static_cast<T>(var_inv) *
                static_cast<T>(var_inv) * s;
    p2[ng] = x;
    p3[ng] = -x * static_cast<T>(mean[ng]) - sum2 * static_cast<T>(var_inv) * s;
  }
}

template <typename T>
__global__ void GetXGradientCUDAKernel(int imsize,
                                       int C,
                                       int group_size,
                                       int groups,
                                       T* p1,
                                       T* p2,
                                       T* p3,
                                       const T* x,
                                       const T* dy,
                                       T* dx) {
  int cid = blockIdx.x;
  int gid = blockIdx.y;
  int bid = blockIdx.z;
  int ccid = bid * C + gid * group_size + cid;
  int ng = bid * groups + gid;
  int nc = gid * group_size + cid;
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    int index = (bid * C + nc) * imsize + imid;
    dx[index] = p1[ccid] * dy[index] + p2[ng] * x[index] + p3[ng];
  }
}

template <typename T>
class GroupNormGradKernel<phi::GPUContext, T> : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const float epsilon = ctx.Attr<float>("epsilon");
    auto* x = ctx.Input<Tensor>("X");
    auto* y = ctx.Input<Tensor>("Y");
    auto* mean = ctx.Input<Tensor>("Mean");
    auto* var = ctx.Input<Tensor>("Variance");
    auto* scale = ctx.Input<Tensor>("Scale");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto groups = ctx.Attr<int>("groups");

    // init output
    auto* d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto* d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    const auto& x_dims = x->dims();
    const int C =
        (data_layout == DataLayout::kNCHW ? x_dims[1]
                                          : x_dims[x_dims.size() - 1]);
    const int group_size = C / groups;
    const int W =
        (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                          : x_dims[x_dims.size() - 2]);

    d_x->mutable_data<T>(ctx.GetPlace());
    phi::funcs::SetConstant<phi::GPUContext, T> set_zero;
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();

    Tensor ds, db;
    ds.mutable_data<T>({x_dims[0], C}, ctx.GetPlace());
    db.mutable_data<T>({x_dims[0], C}, ctx.GetPlace());
    T* ds_data = ds.data<T>();
    T* db_data = db.data<T>();

    auto* y_data = y->data<T>();
    auto* x_data = x->data<T>();
    T* d_x_data = nullptr;
    if (d_x) d_x_data = d_x->data<T>();
    auto* dy_data = d_y->data<T>();
    auto* var_data = var->data<T>();
    auto* mean_data = mean->data<T>();
    T* d_scale_data = nullptr;
    if (d_scale) {
      d_scale->mutable_data<T>(ctx.GetPlace());
      d_scale_data = d_scale->data<T>();
    }
    T* d_bias_data = nullptr;
    if (d_bias) {
      d_bias->mutable_data<T>(ctx.GetPlace());
      d_bias_data = d_bias->data<T>();
    }

    const T* scale_data = nullptr;
    if (scale) scale_data = scale->data<T>();
    const T* bias_data = nullptr;
    if (bias) bias_data = bias->data<T>();

    int imsize = 1;
    if (data_layout == DataLayout::kNCHW) {
      for (int i = 2; i < x_dims.size(); ++i) {
        imsize *= x_dims[i];
      }
    } else {
      for (int i = 1; i < x_dims.size() - 1; ++i) {
        imsize *= x_dims[i];
      }
    }

#ifdef __HIPCC__
    int block_size = std::max(std::min(256, imsize), 64);
    const int block_dims = 256;
#else
    int block_size = std::min(1024, imsize);
    const int block_dims = 1024;
#endif
    dim3 grid(group_size, groups, x_dims[0]);
    dim3 threads(block_size, 1, 1);
    int flags =
        (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
    if (data_layout == DataLayout::kNCHW) {
      const int max_num_threads = 1024;
      int max_block_size = std::min(imsize, max_num_threads);
      int block_size_nchw = 1;
      while (block_size_nchw < max_block_size) {
        block_size_nchw *= 2;
      }
      block_size_nchw = std::max(block_size_nchw, kps::details::kWarpSize);
      dim3 blocks(block_size_nchw);
      ScalarGetDsDbCUDAKernel<T>
          <<<x_dims[0] * C, blocks, 0, dev_ctx.stream()>>>(
              imsize, x_data, dy_data, ds_data, db_data);

      if (d_scale || d_bias) {
        const int block = 256;
        GetScaleBiasGradientCUDAKernel<T>
            <<<(C + block - 1) / block, block, 0, dev_ctx.stream()>>>(
                x_dims[0],
                C,
                groups,
                epsilon,
                mean_data,
                var_data,
                ds_data,
                db_data,
                d_scale_data,
                d_bias_data);
      }

      if (d_x_data != nullptr) {
        // p1 * dy + p2 * x + p3,
        // p1, p2, p3 represent the reverse calculation of temporary variables
        // p1 = scale * var_inv
        // p2 = (db * scale * mean - ds * scale) * pow(var_inv, 3) * (1/n)
        // p3 = -p2 * mean[ng] - db * scale * var_inv * (1/n);
        Tensor p1, p2, p3;
        p1.mutable_data<T>({x_dims[0] * C}, ctx.GetPlace());
        p2.mutable_data<T>({x_dims[0], groups}, ctx.GetPlace());
        p3.mutable_data<T>({x_dims[0], groups}, ctx.GetPlace());
        T* p1_data = p1.data<T>();
        T* p2_data = p2.data<T>();
        T* p3_data = p3.data<T>();

        GetBackwardParamsCUDAKernel<T, block_dims>
            <<<dim3(x_dims[0], groups), block_dims, 0, dev_ctx.stream()>>>(
                imsize,
                groups,
                group_size,
                epsilon,
                mean_data,
                var_data,
                scale_data,
                ds_data,
                db_data,
                p1_data,
                p2_data,
                p3_data);
        GetXGradientCUDAKernel<T>
            <<<grid, threads, 0, dev_ctx.stream()>>>(imsize,
                                                     C,
                                                     group_size,
                                                     groups,
                                                     p1_data,
                                                     p2_data,
                                                     p3_data,
                                                     x_data,
                                                     dy_data,
                                                     d_x_data);
      }
    } else {
      if (d_scale) {
        set_zero(dev_ctx, d_scale, static_cast<T>(0));
      }
      if (d_bias) {
        set_zero(dev_ctx, d_bias, static_cast<T>(0));
      }

      Tensor temp_var;
      temp_var.mutable_data<T>(var->dims(), ctx.GetPlace());
      set_zero(dev_ctx, &temp_var, static_cast<T>(0));
      T* temp_var_data = temp_var.data<T>();

      Tensor temp_mean;
      temp_mean.mutable_data<T>(var->dims(), ctx.GetPlace());
      set_zero(dev_ctx, &temp_mean, static_cast<T>(0));
      T* temp_mean_data = temp_mean.data<T>();

      int flags = (scale_data != nullptr) * kHasScale +
                  (bias_data != nullptr) * kHasBias;
      UNROLL_ALL_CASES(flags,
                       GroupNormBackwardGetMeanAndVar,
                       y_data,
                       scale_data,
                       bias_data,
                       dy_data,
                       x_dims[0],
                       C,
                       W,
                       imsize,
                       groups,
                       group_size,
                       epsilon,
                       temp_mean_data,
                       temp_var_data,
                       d_scale_data,
                       d_bias_data);
      if (d_x_data != nullptr) {
        UNROLL_ALL_CASES(flags,
                         GroupNormBackward,
                         y_data,
                         dy_data,
                         scale_data,
                         bias_data,
                         var_data,
                         temp_mean_data,
                         temp_var_data,
                         x_dims[0],
                         C,
                         W,
                         imsize,
                         groups,
                         group_size,
                         epsilon,
                         d_x_data);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(group_norm,
                        ops::GroupNormKernel<phi::GPUContext, float>,
                        ops::GroupNormKernel<phi::GPUContext, double>);
REGISTER_OP_CUDA_KERNEL(group_norm_grad,
                        ops::GroupNormGradKernel<phi::GPUContext, float>,
                        ops::GroupNormGradKernel<phi::GPUContext, double>);
