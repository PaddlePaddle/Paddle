#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "hipcub/hipcub.hpp"
#include "paddle/fluid/operators/group_norm_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"

namespace paddle {
namespace operators {

using DataLayout = framework::DataLayout;
enum GroupNormKernelFlags { kHasScale = 1, kHasBias = 2 };

#define CHECK_CASE(i, flags, kernel_name, ...)                              \
  if (i == flags) {                                                         \
    kernel_name<T, i><<<grid, threads, 0, dev_ctx.stream()>>>(__VA_ARGS__); \
  }

// 0 for no scale, no bias
// 1 for has scale, no bias
// 2 for no scale, has bias
// 3 for has scale, has bias
#define UNROLL_ALL_CASES(flags, kernel_name, ...) \
  CHECK_CASE(0, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(1, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(2, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(3, flags, kernel_name, __VA_ARGS__)

template <typename T>
__device__ __inline__ void CudaAtomicAddWithWarp(T* sum, T value) {
  typedef hipcub::WarpReduce<T> WarpReduce;
  typename WarpReduce::TempStorage temp_storage;
  value = WarpReduce(temp_storage).Sum(value);
  if (hipcub::LaneId() == 0) platform::CudaAtomicAdd(sum, value);
}

template <typename T>
__global__ void GroupNormForwardGetMeanAndVar(const T* x, int N, int C, int W,
                                              int imsize, int groups,
                                              int group_size, T* mean, T* var,
                                              const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_mean = 0, x_var = 0;
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    if (data_layout == DataLayout::kNCHW) {
      val = x[(bid * C + ccid) * imsize + imid];
    } else {
      int hid = imid / W;
      int wid = imid % W;
      val = x[(bid * H + hid) * W * C + wid * C + ccid];
    }
    x_mean += val;
    x_var += val * val;
  }
  x_mean /= number * imsize;
  x_var /= number * imsize;
  CudaAtomicAddWithWarp(&mean[bid * groups + gid], x_mean);
  CudaAtomicAddWithWarp(&var[bid * groups + gid], x_var);
}

template <typename T, int flags>
__global__ void GroupNormForward(const T* x, const T* mean, const T* var,
                                 const T* scale, const T* bias, int N, int C,
                                 int W, int imsize, int groups, int group_size,
                                 T epsilon, T* y, T* real_var,
                                 const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_mean = mean[bid * groups + gid];
  T x_var = var[bid * groups + gid];
  x_var = x_var - x_mean * x_mean;
  T var_inv = 1.0 / sqrt(x_var + epsilon);
  if (cid == 0 && threadIdx.x == 0) real_var[bid * groups + gid] = x_var;
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    int hid, wid;
    if (data_layout == DataLayout::kNCHW) {
      val = x[(bid * C + ccid) * imsize + imid];
    } else {
      hid = imid / W;
      wid = imid % W;
      val = x[(bid * H + hid) * W * C + wid * C + ccid];
    }
    val = (val - x_mean) * var_inv;
    if (flags & kHasScale) val *= scale[gid * group_size + cid];
    if (flags & kHasBias) val += bias[gid * group_size + cid];
    if (data_layout == DataLayout::kNCHW) {
      y[(bid * C + ccid) * imsize + imid] = val;
    } else {
      y[(bid * H + hid) * W * C + wid * C + ccid] = val;
    }
  }
}

template <typename T>
class GroupNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const float epsilon = ctx.Attr<float>("epsilon");
    auto* scale = ctx.Input<Tensor>("Scale");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* x = ctx.Input<Tensor>("X");

    auto* y = ctx.Output<Tensor>("Y");
    auto* mean = ctx.Output<Tensor>("Mean");
    auto* var = ctx.Output<Tensor>("Variance");
    const auto groups = ctx.Attr<int>("groups");

    const auto x_dims = x->dims();
    const int C =
        (data_layout == DataLayout::kNCHW ? x_dims[1]
                                          : x_dims[x_dims.size() - 1]);
    const int group_size = (C - 1) / groups + 1;
    const int W =
        (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                          : x_dims[x_dims.size() - 2]);

    y->mutable_data<T>(ctx.GetPlace());
    mean->mutable_data<T>(ctx.GetPlace());
    var->mutable_data<T>(ctx.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    Tensor temp_var;
    temp_var.mutable_data<T>(var->dims(), ctx.GetPlace());

    set_zero(dev_ctx, mean, static_cast<T>(0));
    set_zero(dev_ctx, &temp_var, static_cast<T>(0));

    auto* x_data = x->data<T>();
    auto* y_data = y->data<T>();
    auto* mean_data = mean->data<T>();
    auto* var_data = var->data<T>();
    auto* temp_var_data = temp_var.data<T>();

    const T* scale_data = nullptr;
    if (scale) scale_data = scale->data<T>();
    const T* bias_data = nullptr;
    if (bias) bias_data = bias->data<T>();

    int imsize = (data_layout == DataLayout::kNCHW ? x_dims[2] * x_dims[3]
                                                   : x_dims[1] * x_dims[2]);

    int block_size = std::min(1024, imsize);
    dim3 grid(group_size, groups, x_dims[0]);
    dim3 threads(block_size, 1, 1);
    GroupNormForwardGetMeanAndVar<T><<<grid, threads, 0, dev_ctx.stream()>>>(
        x_data, x_dims[0], C, W, imsize, groups, group_size, mean_data,
        temp_var_data, data_layout);
    int flags =
        (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
    UNROLL_ALL_CASES(flags, GroupNormForward, x_data, mean_data, temp_var_data,
                     scale_data, bias_data, x_dims[0], C, W, imsize, groups,
                     group_size, epsilon, y_data, var_data, data_layout);
  }
};

template <typename T, int flags>
__global__ void GroupNormBackwardGetMeanAndVar(
    const T* x, const T* scale, const T* bias, const T* d_y, int N, int C,
    int W, int imsize, int groups, int group_size, T epsilon, T* d_mean,
    T* d_var, T* d_scale, T* d_bias, const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_scale = (flags & kHasScale) ? scale[ccid] : 1;
  T x_bias = (flags & kHasBias) ? bias[ccid] : 0;
  T x_scale_inv = 0;
  if (x_scale != 0) x_scale_inv = 1.0 / x_scale;
  T d_mean_data = 0, d_var_data = 0, d_scale_data = 0, d_bias_data = 0;

  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val, dval;
    if (data_layout == DataLayout::kNCHW) {
      val = x[(bid * C + ccid) * imsize + imid] - x_bias;
      dval = d_y[(bid * C + ccid) * imsize + imid];
    } else {
      int hid = imid / W;
      int wid = imid % W;
      val = x[(bid * H + hid) * W * C + wid * C + ccid] - x_bias;
      dval = d_y[(bid * H + hid) * W * C + wid * C + ccid];
    }

    d_var_data += val * dval;
    d_mean_data += dval * x_scale;

    val = val * x_scale_inv;
    d_bias_data += dval;
    d_scale_data += val * dval;
  }
  CudaAtomicAddWithWarp(&d_mean[bid * groups + gid], d_mean_data);
  CudaAtomicAddWithWarp(&d_var[bid * groups + gid], d_var_data);
  if (flags & kHasScale) CudaAtomicAddWithWarp(&d_scale[ccid], d_scale_data);
  if (flags & kHasBias) CudaAtomicAddWithWarp(&d_bias[ccid], d_bias_data);
}

template <typename T, int flags>
__global__ void GroupNormBackward(const T* x, const T* d_y, const T* scale,
                                  const T* bias, const T* var, const T* d_mean,
                                  const T* d_var, int N, int C, int W,
                                  int imsize, int groups, int group_size,
                                  T epsilon, T* d_x,
                                  const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_var = var[bid * groups + gid];
  T d_x_mean = d_mean[bid * groups + gid];
  T d_x_var = d_var[bid * groups + gid];

  T x_var_inv = 1.0 / sqrt(x_var + epsilon);
  T number_inv = 1.0 / (number * imsize);

  T x_scale = (flags & kHasScale) ? scale[ccid] : 1;
  T x_bias = (flags & kHasBias) ? bias[ccid] : 0;
  T x_scale_inv = 0;
  if (x_scale != 0) x_scale_inv = 1.0 / x_scale;

  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    if (data_layout == DataLayout::kNCHW) {
      T tmp = x[(bid * C + ccid) * imsize + imid];
      T v_y = (tmp - x_bias) * x_scale_inv;
      T dly = d_y[(bid * C + ccid) * imsize + imid];
      d_x[(bid * C + ccid) * imsize + imid] =
          x_var_inv *
          (dly * x_scale - number_inv * d_x_var * v_y - number_inv * d_x_mean);
    } else {
      int hid = imid / W;
      int wid = imid % W;
      T tmp = x[(bid * H + hid) * W * C + wid * C + ccid];
      T v_y = (tmp - x_bias) * x_scale_inv;
      T dly = d_y[(bid * H + hid) * W * C + wid * C + ccid];
      d_x[(bid * H + hid) * W * C + wid * C + ccid] =
          x_var_inv *
          (dly * x_scale - number_inv * d_x_var * v_y - number_inv * d_x_mean);
    }
  }
}

template <typename T>
class GroupNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const float epsilon = ctx.Attr<float>("epsilon");
    auto* x = ctx.Input<Tensor>("Y");
    auto* var = ctx.Input<Tensor>("Variance");
    auto* scale = ctx.Input<Tensor>("Scale");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto groups = ctx.Attr<int>("groups");

    // init output
    auto* d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto* d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    const auto& x_dims = x->dims();
    const int C =
        (data_layout == DataLayout::kNCHW ? x_dims[1]
                                          : x_dims[x_dims.size() - 1]);
    const int group_size = (C - 1) / groups + 1;
    const int W =
        (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                          : x_dims[x_dims.size() - 2]);

    d_x->mutable_data<T>(ctx.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    Tensor temp_var;
    temp_var.mutable_data<T>(var->dims(), ctx.GetPlace());
    set_zero(dev_ctx, &temp_var, static_cast<T>(0));
    T* temp_var_data = temp_var.data<T>();

    Tensor temp_mean;
    temp_mean.mutable_data<T>(var->dims(), ctx.GetPlace());
    set_zero(dev_ctx, &temp_mean, static_cast<T>(0));
    T* temp_mean_data = temp_mean.data<T>();

    auto* x_data = x->data<T>();
    T* d_x_data = nullptr;
    if (d_x) d_x_data = d_x->data<T>();
    auto* y_data = d_y->data<T>();
    auto* var_data = var->data<T>();
    T* d_scale_data = nullptr;
    if (d_scale) {
      d_scale->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, d_scale, static_cast<T>(0));
      d_scale_data = d_scale->data<T>();
    }
    T* d_bias_data = nullptr;
    if (d_bias) {
      d_bias->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, d_bias, static_cast<T>(0));
      d_bias_data = d_bias->data<T>();
    }

    const T* scale_data = nullptr;
    if (scale) scale_data = scale->data<T>();
    const T* bias_data = nullptr;
    if (bias) bias_data = bias->data<T>();

    int imsize = (data_layout == DataLayout::kNCHW ? x_dims[2] * x_dims[3]
                                                   : x_dims[1] * x_dims[2]);

    int block_size = std::min(1024, imsize);
    dim3 grid(group_size, groups, x_dims[0]);
    dim3 threads(block_size, 1, 1);
    int flags =
        (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
    UNROLL_ALL_CASES(flags, GroupNormBackwardGetMeanAndVar, x_data, scale_data,
                     bias_data, y_data, x_dims[0], C, W, imsize, groups,
                     group_size, epsilon, temp_mean_data, temp_var_data,
                     d_scale_data, d_bias_data, data_layout);
    if (d_x_data != nullptr) {
      UNROLL_ALL_CASES(flags, GroupNormBackward, x_data, y_data, scale_data,
                       bias_data, var_data, temp_mean_data, temp_var_data,
                       x_dims[0], C, W, imsize, groups, group_size, epsilon,
                       d_x_data, data_layout);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    group_norm,
    ops::GroupNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GroupNormKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    group_norm_grad,
    ops::GroupNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GroupNormGradKernel<paddle::platform::CUDADeviceContext, double>);
