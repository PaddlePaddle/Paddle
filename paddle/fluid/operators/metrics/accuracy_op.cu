#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include "paddle/fluid/operators/metrics/accuracy_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;

template <int BlockSize>
__global__ void AccuracyCudaKernel(const int N, const int D,
                                   const int64_t* Xdata,
                                   const int64_t* labeldata, int* correct_data,
                                   float* accuracy, int* total_data) {
  int count = 0;
  __shared__ int total[BlockSize];

  // support only 1 block
  for (int i = threadIdx.x; i < (N); i += BlockSize) {
    for (int j = 0; j < D; ++j) {
      if (Xdata[i * D + j] == labeldata[i]) {
        ++count;
        break;
      }
    }
  }
  total[threadIdx.x] = count;
  __syncthreads();

// reduce the count with init value 0, and output accuracy.
#ifdef PADDLE_WITH_CUDA
  int result = thrust::reduce(thrust::device, total, total + BlockSize, 0);
#else
  // HIP thrust::reduce not support __device__
  for (int s = BlockSize / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      total[threadIdx.x] += total[threadIdx.x + s];
    }
    __syncthreads();
  }
  int result = total[0];
#endif
  if (threadIdx.x == 0) {
    *correct_data = result;
    *accuracy = static_cast<float>(result) / static_cast<float>(N);
    *total_data = N;
  }
}

template <typename T>
class AccuracyOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* inference = ctx.Input<Tensor>("Out");
    auto* indices = ctx.Input<Tensor>("Indices");
    auto* label = ctx.Input<Tensor>("Label");

    auto* accuracy = ctx.Output<Tensor>("Accuracy");
    auto* correct = ctx.Output<Tensor>("Correct");
    auto* total = ctx.Output<Tensor>("Total");
    // FIXME(typhoonzero): only support indices currently
    // if add support for output values, how to detect the data type?
    const int64_t* indices_data = indices->data<int64_t>();
    const int64_t* label_data = label->data<int64_t>();

    int* correct_data = correct->mutable_data<int>(ctx.GetPlace());
    int* total_data = total->mutable_data<int>(ctx.GetPlace());
    float* accuracy_data = accuracy->mutable_data<float>(ctx.GetPlace());

    int num_samples = static_cast<int>(inference->dims()[0]);
    size_t infer_width = inference->dims()[1];
    auto stream = ctx.cuda_device_context().stream();
    platform::GpuMemsetAsync(accuracy_data, 0, sizeof(float), stream);

    if (num_samples == 0) {
      return;
    }

    AccuracyCudaKernel<
        PADDLE_CUDA_NUM_THREADS><<<1, PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        num_samples, infer_width, indices_data, label_data, correct_data,
        accuracy_data, total_data);
  }
};

}  // namespace operators
}  // namespace paddle

// FIXME(typhoonzero): types of T is for inference data.
// label data is always int64
REGISTER_OP_CUDA_KERNEL(
    accuracy, paddle::operators::AccuracyOpCUDAKernel<float>,
    paddle::operators::AccuracyOpCUDAKernel<double>,
    paddle::operators::AccuracyOpCUDAKernel<paddle::platform::float16>);
