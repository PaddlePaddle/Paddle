#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/metrics/auc_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

__global__ void ClearObsoleteDataKernel(int64_t *pos, int64_t *neg,
                                        const int bucket_length,
                                        const int slide_steps) {
  int cur_step_index =
      static_cast<int>(pos[(slide_steps + 1) * bucket_length]) % slide_steps;
  int cur_step_begin = cur_step_index * bucket_length;
  int sum_step_begin = slide_steps * bucket_length;
  CUDA_KERNEL_LOOP(i, bucket_length) {
    pos[sum_step_begin + i] -= pos[cur_step_begin + i];
    neg[sum_step_begin + i] -= neg[cur_step_begin + i];
    pos[cur_step_begin + i] = neg[cur_step_begin + i] = 0;
  }
}

__global__ void UpdateSumDataKernel(int64_t *pos, int64_t *neg,
                                    const int bucket_length,
                                    const int slide_steps) {
  int cur_step_index =
      static_cast<int>(pos[(slide_steps + 1) * bucket_length]) % slide_steps;
  int cur_step_begin = cur_step_index * bucket_length;
  int sum_step_begin = slide_steps * bucket_length;
  CUDA_KERNEL_LOOP(i, bucket_length) {
    pos[sum_step_begin + i] += pos[cur_step_begin + i];
    neg[sum_step_begin + i] += neg[cur_step_begin + i];
  }
}

template <typename T>
__global__ void AddDataKernel(const int64_t *label_data, const T *pred_data,
                              const int inference_width,
                              const int num_thresholds, int64_t *pos,
                              int64_t *neg, const int numel,
                              const int slide_steps) {
  int cur_step_begin = 0;
  if (slide_steps > 0) {
    int cur_step_index =
        static_cast<int>(pos[(slide_steps + 1) * (1 + num_thresholds)]) %
        slide_steps;
    cur_step_begin = cur_step_index * (1 + num_thresholds);
  }
  CUDA_KERNEL_LOOP(i, numel) {
    auto predict_data = pred_data[i * inference_width + (inference_width - 1)];
    PADDLE_ENFORCE(predict_data <= 1, "The predict data must less or equal 1.");
    PADDLE_ENFORCE(predict_data >= 0,
                   "The predict data must gather or equal 0.");
    uint32_t binIdx = static_cast<uint32_t>(predict_data * num_thresholds);
    if (label_data[i]) {
      paddle::platform::CudaAtomicAdd(pos + cur_step_begin + binIdx, 1);
    } else {
      paddle::platform::CudaAtomicAdd(neg + cur_step_begin + binIdx, 1);
    }
  }
}
__global__ void CalcAucKernel(int64_t *stat_pos, int64_t *stat_neg,
                              int num_thresholds, double *auc,
                              bool need_add_batch_num) {
  *auc = 0.0f;
  double totPos = 0.0;
  double totNeg = 0.0;
  double totPosPrev = 0.0;
  double totNegPrev = 0.0;

  int idx = num_thresholds;

  while (idx >= 0) {
    totPosPrev = totPos;
    totNegPrev = totNeg;
    totPos += stat_pos[idx];
    totNeg += stat_neg[idx];
    *auc += (totNeg - totNegPrev) * (totPos + totPosPrev) / 2.0;
    --idx;
  }

  if (totPos > 0.0 && totNeg > 0.0) {
    *auc = *auc / totPos / totNeg;
  }
  if (need_add_batch_num) {
    stat_pos[num_thresholds + 1] += 1;
    stat_neg[num_thresholds + 1] += 1;
  }
}

template <typename DeviceContext, typename T>
class AucCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *predict = ctx.Input<Tensor>("Predict");
    auto *label = ctx.Input<Tensor>("Label");

    int num_thresholds = ctx.Attr<int>("num_thresholds");
    int slide_steps = ctx.Attr<int>("slide_steps");

    // Only use output var for now, make sure it's persistable and
    // not cleaned up for each batch.
    auto *auc_tensor = ctx.Output<Tensor>("AUC");
    auto *stat_pos = ctx.Output<Tensor>("StatPosOut");
    auto *stat_neg = ctx.Output<Tensor>("StatNegOut");

    auto *origin_stat_pos = stat_pos->mutable_data<int64_t>(ctx.GetPlace());
    auto *origin_stat_neg = stat_neg->mutable_data<int64_t>(ctx.GetPlace());
    auto *auc_value = auc_tensor->mutable_data<double>(ctx.GetPlace());

    auto *stat_pos_in_tensor = ctx.Input<Tensor>("StatPos");
    auto *pos_in_data = stat_pos_in_tensor->data<int64_t>();
    auto *stat_neg_in_tensor = ctx.Input<Tensor>("StatNeg");
    auto *neg_in_data = stat_neg_in_tensor->data<int64_t>();
#ifdef PADDLE_WITH_CUDA
    if (stat_pos_in_tensor != stat_pos) {
      hipMemcpy(origin_stat_pos, pos_in_data,
                 ((1 + slide_steps) * (num_thresholds + 1) +
                  (slide_steps > 0 ? 1 : 0)) *
                     sizeof(int64_t),
                 hipMemcpyDeviceToDevice);
    }
    if (stat_neg_in_tensor != stat_neg) {
      hipMemcpy(origin_stat_neg, neg_in_data,
                 ((1 + slide_steps) * (num_thresholds + 1) +
                  (slide_steps > 0 ? 1 : 0)) *
                     sizeof(int64_t),
                 hipMemcpyDeviceToDevice);
    }
#else
    if (stat_pos_in_tensor != stat_pos) {
      hipMemcpy(origin_stat_pos, pos_in_data,
                ((1 + slide_steps) * (num_thresholds + 1) +
                 (slide_steps > 0 ? 1 : 0)) *
                    sizeof(int64_t),
                hipMemcpyDeviceToDevice);
    }
    if (stat_neg_in_tensor != stat_neg) {
      hipMemcpy(origin_stat_neg, neg_in_data,
                ((1 + slide_steps) * (num_thresholds + 1) +
                 (slide_steps > 0 ? 1 : 0)) *
                    sizeof(int64_t),
                hipMemcpyDeviceToDevice);
    }
#endif

    statAuc(ctx, label, predict, num_thresholds, slide_steps, origin_stat_pos,
            origin_stat_neg);
    int sum_offset = slide_steps * (num_thresholds + 1);
    auto stream =
        ctx.template device_context<platform::CUDADeviceContext>().stream();
    CalcAucKernel<<<1, 1, 0, stream>>>(
        origin_stat_pos + sum_offset, origin_stat_neg + sum_offset,
        num_thresholds, auc_value, slide_steps > 0);
  }

 private:
  inline static double trapezoidArea(double X1, double X2, double Y1,
                                     double Y2) {
    return (X1 > X2 ? (X1 - X2) : (X2 - X1)) * (Y1 + Y2) / 2.0;
  }

  inline static void statAuc(const framework::ExecutionContext &ctx,
                             const framework::Tensor *label,
                             const framework::Tensor *predict,
                             const int num_thresholds, const int slide_steps,
                             int64_t *origin_stat_pos,
                             int64_t *origin_stat_neg) {
    size_t batch_size = predict->dims()[0];
    size_t inference_width = predict->dims()[1];
    const T *inference_data = predict->data<T>();
    const auto *label_data = label->data<int64_t>();
    const int bucket_length = num_thresholds + 1;
    auto stream =
        ctx.template device_context<platform::CUDADeviceContext>().stream();
    if (slide_steps == 0) {
      AddDataKernel<<<(batch_size + PADDLE_CUDA_NUM_THREADS - 1) /
                          PADDLE_CUDA_NUM_THREADS,
                      PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          label_data, inference_data, inference_width, num_thresholds,
          origin_stat_pos, origin_stat_neg, batch_size, slide_steps);
      return;
    }
    // the last number of origin_stat_pos store the index should be used in
    // current step
    int cur_step_index =
        static_cast<int>(origin_stat_pos[(slide_steps + 1) * bucket_length]) %
        slide_steps;
    int cur_step_begin = cur_step_index * bucket_length;
    int sum_step_begin = slide_steps * bucket_length;

    ClearObsoleteDataKernel<<<(bucket_length + PADDLE_CUDA_NUM_THREADS - 1) /
                                  PADDLE_CUDA_NUM_THREADS,
                              PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        origin_stat_pos, origin_stat_neg, bucket_length, slide_steps);

    AddDataKernel<<<(batch_size + PADDLE_CUDA_NUM_THREADS - 1) /
                        PADDLE_CUDA_NUM_THREADS,
                    PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        label_data, inference_data, inference_width, num_thresholds,
        origin_stat_pos, origin_stat_neg, batch_size, slide_steps);
    UpdateSumDataKernel<<<(bucket_length + PADDLE_CUDA_NUM_THREADS - 1) /
                              PADDLE_CUDA_NUM_THREADS,
                          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        origin_stat_pos, origin_stat_neg, bucket_length, slide_steps);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(auc,
                        ops::AucCUDAKernel<paddle::platform::CUDAPlace, float>);
