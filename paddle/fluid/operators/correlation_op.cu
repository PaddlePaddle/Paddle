#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifndef PADDLE_WITH_HIP
// HIP not supported yet

#include <algorithm>
#include <string>
#include "paddle/fluid/framework/op_registry.h"

namespace paddle {
namespace operators {

#define THREADS_PER_BLOCK 32
#define FULL_MASK 0xffffffff

using framework::Tensor;

template <typename T>
__forceinline__ __device__ T warpReduceSum(T val) {
  for (int offset = 16; offset > 0; offset /= 2) {
    val += __shfl_down_sync(FULL_MASK, val, offset);
  }
  return val;
}

template <typename T>
__forceinline__ __device__ T blockReduceSum(T val) {
  static __shared__ T shared[32];
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);
  if (lane == 0) shared[wid] = val;

  __syncthreads();
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid == 0) val = warpReduceSum(val);

  return val;
}

template <typename T>
__global__ void set_zero(T *x, int num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x)
    x[i] = static_cast<T>(0);
}

template <typename T>
__global__ void channel_first(const T *input, T *rinput, const int channel,
                              const int height, const int width,
                              const int pad_size) {
  int n = blockIdx.x;
  int h = blockIdx.y;
  int w = blockIdx.z;

  int ch_off = threadIdx.x;
  T value;
  int dimchw = channel * height * width;
  int dimhw = height * width;

  int p_dimw = (width + 2 * pad_size);
  int p_dimh = (height + 2 * pad_size);
  int p_dimchw = channel * p_dimw * p_dimh;
  int p_dimcw = channel * p_dimw;

  for (int c = ch_off; c < channel; c += THREADS_PER_BLOCK) {
    value = input[n * dimchw + c * dimhw + h * width + w];
    rinput[n * p_dimchw + (h + pad_size) * p_dimcw + (w + pad_size) * channel +
           c] = value;
  }
}

template <typename T>
__global__ void correlation_forward(
    T *output, const int output_channel, const int output_height,
    const int output_width, const T *rinput1, const int input_channel,
    const int input_height, const int input_width, const T *rinput2,
    const int pad_size, const int kernel_size, const int max_displacement,
    const int stride1, const int stride2) {
  int p_input_width = input_width + 2 * pad_size;
  int p_input_height = input_height + 2 * pad_size;

  int kernel_rad = (kernel_size - 1) / 2;
  int displacement_rad = max_displacement / stride2;

  int displacement_size = 2 * displacement_rad + 1;

  int n = blockIdx.x;
  int h1 = blockIdx.y * stride1 + max_displacement;
  int w1 = blockIdx.z * stride1 + max_displacement;
  int c = threadIdx.x;

  int p_dimchw = p_input_height * p_input_width * input_channel;
  int p_dimcw = p_input_width * input_channel;
  int p_dimc = input_channel;

  int t_dimchw = output_channel * output_height * output_width;
  int t_dimhw = output_height * output_width;
  int t_dimw = output_width;

  int nelems = kernel_size * kernel_size * p_dimc;

  for (int tj = -displacement_rad; tj <= displacement_rad; ++tj) {
    for (int ti = -displacement_rad; ti <= displacement_rad; ++ti) {
      int w2 = w1 + ti * stride2;
      int h2 = h1 + tj * stride2;

      T acc0 = 0;
      for (int j = -kernel_rad; j <= kernel_rad; ++j) {
        for (int i = -kernel_rad; i <= kernel_rad; ++i) {
          for (int ch = c; ch < p_dimc; ch += blockDim.x) {
            int index1 =
                n * p_dimchw + (h1 + j) * p_dimcw + (w1 + i) * p_dimc + ch;
            int index2 =
                n * p_dimchw + (h2 + j) * p_dimcw + (w2 + i) * p_dimc + ch;
            acc0 += static_cast<T>(rinput1[index1] * rinput2[index2]);
          }
        }
      }
      if (blockDim.x == warpSize) {
        __syncwarp();
        acc0 = warpReduceSum(acc0);
      } else {
        __syncthreads();
        acc0 = blockReduceSum(acc0);
      }

      if (threadIdx.x == 0) {
        int tc = (tj + displacement_rad) * displacement_size +
                 (ti + displacement_rad);
        const int t_index =
            n * t_dimchw + tc * t_dimhw + blockIdx.y * t_dimw + blockIdx.z;
        output[t_index] = static_cast<T>(acc0 / nelems);
      }
    }
  }
}

// class CorrelationKernel<platform::CUDADeviceContext, T>
template <typename T>
class CorrelationCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      platform::errors::InvalidArgument(
                          "Correlation only supports GPU now."));

    auto *input1 = ctx.Input<Tensor>("Input1");
    auto *input2 = ctx.Input<Tensor>("Input2");
    int pad_size = ctx.Attr<int>("pad_size");
    int kernel_size = ctx.Attr<int>("kernel_size");
    int stride1 = ctx.Attr<int>("stride1");
    int stride2 = ctx.Attr<int>("stride2");
    int max_displacement = ctx.Attr<int>("max_displacement");
    int corr_type_multiply = ctx.Attr<int>("corr_type_multiply");

    auto *output = ctx.Output<Tensor>("Output");
    output->mutable_data<T>(ctx.GetPlace());
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    // base on input1, NCHW
    auto in_dims = input1->dims();
    int N = in_dims[0];
    int C = in_dims[1];
    int H = in_dims[2];
    int W = in_dims[3];

    int padded_input_height = H + 2 * pad_size;
    int padded_input_width = W + 2 * pad_size;

    Tensor rinput1 = ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>(
        {N, padded_input_height, padded_input_width, C}, dev_ctx);
    rinput1.mutable_data<T>(ctx.GetPlace());

    Tensor rinput2 = ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>(
        {N, padded_input_height, padded_input_width, C}, dev_ctx);
    rinput2.mutable_data<T>(ctx.GetPlace());

    set_zero<<<(rinput1.numel() + 512 - 1) / 512, 512, 0, dev_ctx.stream()>>>(
        rinput1.data<T>(), rinput1.numel());
    set_zero<<<(rinput2.numel() + 512 - 1) / 512, 512, 0, dev_ctx.stream()>>>(
        rinput2.data<T>(), rinput2.numel());
    set_zero<<<(output->numel() + 512 - 1) / 512, 512, 0, dev_ctx.stream()>>>(
        output->data<T>(), output->numel());

    auto out_dims = output->dims();
    int OC = out_dims[1];
    int OH = out_dims[2];
    int OW = out_dims[3];

    dim3 blocks_grid(N, H, W);
    dim3 threads_block(THREADS_PER_BLOCK);

    channel_first<T><<<blocks_grid, threads_block, 0, dev_ctx.stream()>>>(
        input1->data<T>(), rinput1.data<T>(), C, H, W, pad_size);
    channel_first<T><<<blocks_grid, threads_block, 0, dev_ctx.stream()>>>(
        input2->data<T>(), rinput2.data<T>(), C, H, W, pad_size);

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 totalBlocksCorr(N, OH, OW);

    correlation_forward<
        T><<<totalBlocksCorr, threadsPerBlock, 0, dev_ctx.stream()>>>(
        output->data<T>(), OC, OH, OW, rinput1.data<T>(), C, H, W,
        rinput2.data<T>(), pad_size, kernel_size, max_displacement, stride1,
        stride2);
  }
};

template <typename T>
__global__ void correlation_backward_input1(
    int item, T *grad_input1, const int input_channel, const int input_height,
    const int input_width, const T *grad_output, const int output_channel,
    const int output_height, const int output_width, const T *rinput2,
    const int pad_size, const int kernel_size, const int max_displacement,
    const int stride1, const int stride2) {
  int n = item;
  int h = blockIdx.x * stride1 + pad_size;
  int w = blockIdx.y * stride1 + pad_size;
  int c = blockIdx.z;
  int tch_off = threadIdx.x;

  int kernel_rad = (kernel_size - 1) / 2;
  int displacement_rad = max_displacement / stride2;
  int displacement_size = 2 * displacement_rad + 1;

  int xmin = (w - kernel_rad - max_displacement) / stride1;
  int ymin = (h - kernel_rad - max_displacement) / stride1;

  int xmax = (w + kernel_rad - max_displacement) / stride1;
  int ymax = (h + kernel_rad - max_displacement) / stride1;

  if (xmax < 0 || ymax < 0 || xmin >= output_width || ymin >= output_height) {
    return;
  }

  if (xmin > xmax || ymin > ymax) {
    return;
  }

  xmin = max(0, xmin);
  xmax = min(output_width - 1, xmax);

  ymin = max(0, ymin);
  ymax = min(output_height - 1, ymax);

  int p_input_width = input_width + 2 * pad_size;
  int p_input_height = input_height + 2 * pad_size;
  int p_dimchw = input_channel * p_input_height * p_input_width;
  int p_dimcw = input_channel * p_input_width;
  int p_dimc = input_channel;

  int t_dimchw = output_channel * output_height * output_width;
  int t_dimhw = output_height * output_width;
  int t_dimw = output_width;

  int o_dimchw = input_channel * input_height * input_width;
  int o_dimhw = input_height * input_width;
  int o_dimw = input_width;

  int nelems = kernel_size * kernel_size * input_channel;

  __shared__ T prod_sum[THREADS_PER_BLOCK];
  prod_sum[tch_off] = 0;

  for (int tc = tch_off; tc < output_channel; tc += THREADS_PER_BLOCK) {
    int i2 = (tc % displacement_size - displacement_rad) * stride2;
    int j2 = (tc / displacement_size - displacement_rad) * stride2;

    int index2 = n * p_dimchw + (h + j2) * p_dimcw + (w + i2) * p_dimc + c;

    T val2 = rinput2[index2];
    for (int j = ymin; j <= ymax; ++j) {
      for (int i = xmin; i <= xmax; ++i) {
        int t_index = n * t_dimchw + tc * t_dimhw + j * t_dimw + i;
        prod_sum[tch_off] += grad_output[t_index] * val2;
      }
    }
  }

  __syncthreads();

  if (tch_off == 0) {
    T reduce_sum = 0;
    for (int index = 0; index < THREADS_PER_BLOCK; index++) {
      reduce_sum += prod_sum[index];
    }
    const int index1 =
        n * o_dimchw + c * o_dimhw + (h - pad_size) * o_dimw + (w - pad_size);
    grad_input1[index1] = static_cast<T>(reduce_sum / nelems);
  }
}

template <typename T>
__global__ void correlation_backward_input2(
    int item, T *grad_input2, const int input_channel, const int input_height,
    const int input_width, const T *grad_output, const int output_channel,
    const int output_height, const int output_width, const T *rinput1,
    const int pad_size, const int kernel_size, const int max_displacement,
    const int stride1, const int stride2) {
  int n = item;
  int h = blockIdx.x * stride1 + pad_size;
  int w = blockIdx.y * stride1 + pad_size;
  int c = blockIdx.z;

  int tch_off = threadIdx.x;

  int kernel_rad = (kernel_size - 1) / 2;
  int displacement_rad = max_displacement / stride2;
  int displacement_size = 2 * displacement_rad + 1;

  int p_input_width = input_width + 2 * pad_size;
  int p_input_height = input_height + 2 * pad_size;
  int p_dimchw = input_channel * p_input_height * p_input_width;
  int p_dimcw = input_channel * p_input_width;
  int p_dimc = input_channel;

  int t_dimchw = output_channel * output_height * output_width;
  int t_dimhw = output_height * output_width;
  int t_dimw = output_width;

  int o_dimchw = input_channel * input_height * input_width;
  int o_dimhw = input_height * input_width;
  int o_dimw = input_width;

  int nelems = kernel_size * kernel_size * input_channel;

  __shared__ T prod_sum[THREADS_PER_BLOCK];
  prod_sum[tch_off] = 0;

  for (int tc = tch_off; tc < output_channel; tc += THREADS_PER_BLOCK) {
    int i2 = (tc % displacement_size - displacement_rad) * stride2;
    int j2 = (tc / displacement_size - displacement_rad) * stride2;

    int xmin = (w - kernel_rad - max_displacement - i2) / stride1;
    int ymin = (h - kernel_rad - max_displacement - j2) / stride1;

    int xmax = (w + kernel_rad - max_displacement - i2) / stride1;
    int ymax = (h + kernel_rad - max_displacement - j2) / stride1;

    if (xmax < 0 || ymax < 0 || xmin >= output_width || ymin >= output_height) {
      continue;
    }

    if (xmin > xmax || ymin > ymax) {
      continue;
    }

    xmin = max(0, xmin);
    xmax = min(output_width - 1, xmax);

    ymin = max(0, ymin);
    ymax = min(output_height - 1, ymax);

    int index1 = n * p_dimchw + (h - j2) * p_dimcw + (w - i2) * p_dimc + c;
    T val1 = rinput1[index1];
    for (int j = ymin; j <= ymax; ++j) {
      for (int i = xmin; i <= xmax; ++i) {
        int t_index = n * t_dimchw + tc * t_dimhw + j * t_dimw + i;
        prod_sum[tch_off] += grad_output[t_index] * val1;
      }
    }
  }

  __syncthreads();

  if (tch_off == 0) {
    T reduce_sum = 0;
    for (int index = 0; index < THREADS_PER_BLOCK; index++) {
      reduce_sum += prod_sum[index];
    }
    const int index2 =
        n * o_dimchw + c * o_dimhw + (h - pad_size) * o_dimw + (w - pad_size);
    grad_input2[index2] = static_cast<T>(reduce_sum / nelems);
  }
}

template <typename T>
class CorrelationCUDAGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      platform::errors::InvalidArgument(
                          "Correlation only supports GPU now."));
    const auto *input1 = ctx.Input<Tensor>("Input1");
    const auto *input2 = ctx.Input<Tensor>("Input2");
    const auto *grad_output =
        ctx.Input<Tensor>(framework::GradVarName("Output"));
    const int pad_size = ctx.Attr<int>("pad_size");
    const int kernel_size = ctx.Attr<int>("kernel_size");
    const int stride1 = ctx.Attr<int>("stride1");
    const int stride2 = ctx.Attr<int>("stride2");
    const int max_displacement = ctx.Attr<int>("max_displacement");
    const int corr_type_multiply = ctx.Attr<int>("corr_type_multiply");

    auto *grad_input1 = ctx.Output<Tensor>(framework::GradVarName("Input1"));
    grad_input1->mutable_data<T>(ctx.GetPlace());
    auto *grad_input2 = ctx.Output<Tensor>(framework::GradVarName("Input2"));
    grad_input2->mutable_data<T>(ctx.GetPlace());
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    auto in_dims = input1->dims();
    int N = in_dims[0];
    int C = in_dims[1];
    int H = in_dims[2];
    int W = in_dims[3];

    int padded_input_height = H + 2 * pad_size;
    int padded_input_width = W + 2 * pad_size;

    Tensor rinput1 = ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>(
        {N, padded_input_height, padded_input_width, C}, dev_ctx);
    rinput1.mutable_data<T>(ctx.GetPlace());

    Tensor rinput2 = ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>(
        {N, padded_input_height, padded_input_width, C}, dev_ctx);
    rinput2.mutable_data<T>(ctx.GetPlace());

    set_zero<<<(rinput1.numel() + 512 - 1) / 512, 512, 0, dev_ctx.stream()>>>(
        rinput1.data<T>(), rinput1.numel());
    set_zero<<<(rinput2.numel() + 512 - 1) / 512, 512, 0, dev_ctx.stream()>>>(
        rinput2.data<T>(), rinput2.numel());
    set_zero<<<(grad_input1->numel() + 512 - 1) / 512, 512, 0,
               dev_ctx.stream()>>>(grad_input1->data<T>(),
                                   grad_input1->numel());
    set_zero<<<(grad_input2->numel() + 512 - 1) / 512, 512, 0,
               dev_ctx.stream()>>>(grad_input2->data<T>(),
                                   grad_input2->numel());

    auto grad_out_dims = grad_output->dims();
    int GOC = grad_out_dims[1];
    int GOH = grad_out_dims[2];
    int GOW = grad_out_dims[3];

    dim3 blocks_grid(N, H, W);
    dim3 threads_block(THREADS_PER_BLOCK);

    channel_first<T><<<blocks_grid, threads_block, 0, dev_ctx.stream()>>>(
        input1->data<T>(), rinput1.data<T>(), C, H, W, pad_size);
    channel_first<T><<<blocks_grid, threads_block, 0, dev_ctx.stream()>>>(
        input2->data<T>(), rinput2.data<T>(), C, H, W, pad_size);

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 totalBlocksCorr(H, W, C);

    for (int n = 0; n < N; n++) {
      correlation_backward_input1<
          T><<<totalBlocksCorr, threadsPerBlock, 0, dev_ctx.stream()>>>(
          n, grad_input1->data<T>(), C, H, W, grad_output->data<T>(), GOC, GOH,
          GOW, rinput2.data<T>(), pad_size, kernel_size, max_displacement,
          stride1, stride2);
    }

    for (int n = 0; n < N; n++) {
      correlation_backward_input2<
          T><<<totalBlocksCorr, threadsPerBlock, 0, dev_ctx.stream()>>>(
          n, grad_input2->data<T>(), C, H, W, grad_output->data<T>(), GOC, GOH,
          GOW, rinput1.data<T>(), pad_size, kernel_size, max_displacement,
          stride1, stride2);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(correlation, ops::CorrelationCUDAKernel<float>,
                        ops::CorrelationCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(correlation_grad, ops::CorrelationCUDAGradKernel<float>,
                        ops::CorrelationCUDAGradKernel<double>);

#endif  // not PADDLE_WITH_HIP
