#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/fake_quantize_op.cu.h"
#include "paddle/fluid/operators/quantize_linear_op.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"

using float16 = paddle::platform::float16;

namespace paddle {
namespace operators {

template <typename T>
__global__ void KeDequantize(
    const T* in, const T* scale, T max_range, int64_t num, T* out) {
  int64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  for (int64_t i = idx; i < num; i += blockDim.x * gridDim.x) {
    out[i] = in[i] * scale[0] / max_range;
  }
}

template <typename T>
__global__ void DequantizeOneScaleQuantAxisN(const T* in,
                                             const T* scale,
                                             const T max_range,
                                             const int64_t num,
                                             const int n_scales,
                                             const int quant_stride,
                                             T* out) {
  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int64_t i = idx; i < num; i += blockDim.x * gridDim.x) {
    T s = scale[(i / quant_stride) % n_scales];
    out[i] = in[i] * s / max_range;
  }
}

template <typename T>
struct DequantizeFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& dev_ctx,
                  const phi::DenseTensor* in,
                  const phi::DenseTensor* scale,
                  T max_range,
                  phi::DenseTensor* out) {
    const T* in_data = in->data<T>();
    const T* scale_factor = scale->data<T>();
    T* out_data = dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));

    int64_t num = in->numel();
    int64_t block_size = std::min(
        num, static_cast<int64_t>(dev_ctx.GetMaxThreadsPerBlock() / 4));
    int64_t max_threads =
        dev_ctx.GetMaxPhysicalThreadCount();  // SM * block_per_SM
    const int64_t max_blocks =
        std::max(((max_threads - 1) / block_size + 1), static_cast<int64_t>(1));
    const int64_t grid_size =
        std::min(max_blocks, (num + block_size - 1) / block_size);
    KeDequantize<T><<<grid_size, block_size, 0, dev_ctx.stream()>>>(
        in_data, scale_factor, max_range, num, out_data);
  }
};

template <typename T>
struct ChannelDequantizeFunctorV2<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& dev_ctx,
                  const phi::DenseTensor* in,
                  const phi::DenseTensor* scale,
                  T max_range,
                  const int quant_axis,
                  phi::DenseTensor* out) {
    auto in_dims = in->dims();
    const T* in_data = in->data<T>();
    T* out_data = dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));
    int64_t num = in->numel();
    const T* scale_factor = scale->data<T>();
    int64_t block_size = std::min(
        num, static_cast<int64_t>(dev_ctx.GetMaxThreadsPerBlock() / 4));
    int64_t max_threads =
        dev_ctx.GetMaxPhysicalThreadCount();  // SM * block_per_SM
    const int64_t max_blocks =
        std::max(((max_threads - 1) / block_size + 1), static_cast<int64_t>(1));
    const int64_t grid_size =
        std::min(max_blocks, (num + block_size - 1) / block_size);

    int quant_stride = 1;
    for (int i = quant_axis + 1; i < in_dims.size(); i++) {
      quant_stride *= in_dims[i];
    }

    DequantizeOneScaleQuantAxisN<T>
        <<<grid_size, block_size, 0, dev_ctx.stream()>>>(in_data,
                                                         scale_factor,
                                                         max_range,
                                                         num,
                                                         in_dims[quant_axis],
                                                         quant_stride,
                                                         out_data);
  }
};

template struct DequantizeFunctor<phi::GPUContext, phi::dtype::float16>;
template struct DequantizeFunctor<phi::GPUContext, float>;
template struct DequantizeFunctor<phi::GPUContext, double>;
template struct ChannelDequantizeFunctorV2<phi::GPUContext, float16>;
template struct ChannelDequantizeFunctorV2<phi::GPUContext, float>;
template struct ChannelDequantizeFunctorV2<phi::GPUContext, double>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = phi::GPUContext;
REGISTER_OP_CUDA_KERNEL(dequantize_linear,
                        ops::DeQuantizeLinearKernel<CUDA, float>,
                        ops::DeQuantizeLinearKernel<CUDA, float16>,
                        ops::DeQuantizeLinearKernel<CUDA, int8_t>,
                        ops::DeQuantizeLinearKernel<CUDA, double>);

REGISTER_OP_CUDA_KERNEL(quantize_linear,
                        ops::QuantizeLinearKernel<CUDA, float>,
                        ops::QuantizeLinearKernel<CUDA, float16>);
