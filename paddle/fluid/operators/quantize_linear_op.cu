#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/quantize_linear_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void KeDequantize(const T* in, const T* scale, T max_range,
                             int64_t num, T* out) {
  int64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  for (int64_t i = idx; i < num; i += blockDim.x * gridDim.x) {
    out[i] = in[i] * scale[0] / max_range;
  }
}

template <typename T>
struct DequantizeFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor* in, const framework::Tensor* scale,
                  T max_range, framework::Tensor* out) {
    const T* in_data = in->data<T>();
    const T* scale_factor = scale->data<T>();
    T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());

    int64_t num = in->numel();
    int64_t block_size = std::min(
        num, static_cast<int64_t>(dev_ctx.GetMaxThreadsPerBlock() / 4));
    int64_t max_threads =
        dev_ctx.GetMaxPhysicalThreadCount();  // SM * block_per_SM
    const int64_t max_blocks =
        std::max(((max_threads - 1) / block_size + 1), static_cast<int64_t>(1));
    const int64_t grid_size =
        std::min(max_blocks, (num + block_size - 1) / block_size);
    KeDequantize<T><<<grid_size, block_size, 0, dev_ctx.stream()>>>(
        in_data, scale_factor, max_range, num, out_data);
  }
};

template <typename T>
__global__ void DequantizeOneScaleQuantAxisN(const T* in, const T* scale,
                                             const T max_range,
                                             const int64_t num,
                                             const int n_scales,
                                             const int quant_stride, T* out) {
  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int64_t i = idx; i < num; i += blockDim.x * gridDim.x) {
    T s = scale[(i / quant_stride) % n_scales];
    out[i] = in[i] * s / max_range;
  }
}

template <typename T>
struct ChannelDequantizeFunctorV2<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor* in, const framework::Tensor* scale,
                  T max_range, const int quant_axis, framework::Tensor* out) {
    auto in_dims = in->dims();
    const T* in_data = in->data<T>();
    T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());
    int64_t num = in->numel();
    const T* scale_factor = scale->data<T>();
    int64_t block_size = std::min(
        num, static_cast<int64_t>(dev_ctx.GetMaxThreadsPerBlock() / 4));
    int64_t max_threads =
        dev_ctx.GetMaxPhysicalThreadCount();  // SM * block_per_SM
    const int64_t max_blocks =
        std::max(((max_threads - 1) / block_size + 1), static_cast<int64_t>(1));
    const int64_t grid_size =
        std::min(max_blocks, (num + block_size - 1) / block_size);

    int quant_stride = 1;
    for (int i = quant_axis + 1; i < in_dims.size(); i++) {
      quant_stride *= in_dims[i];
    }

    DequantizeOneScaleQuantAxisN<
        T><<<grid_size, block_size, 0, dev_ctx.stream()>>>(
        in_data, scale_factor, max_range, num, in_dims[quant_axis],
        quant_stride, out_data);
  }
};

template struct DequantizeFunctor<platform::CUDADeviceContext, float>;
template struct DequantizeFunctor<platform::CUDADeviceContext, double>;
template struct ChannelDequantizeFunctorV2<platform::CUDADeviceContext, float>;
template struct ChannelDequantizeFunctorV2<platform::CUDADeviceContext, double>;

template <typename T>
__global__ void FindAbsMaxKernel(const T* in, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ char* shared_max_data_tmp[];
  auto shared_max_data = reinterpret_cast<T*>(shared_max_data_tmp);
  if (gridDim.x > 1) {
    T local_max_data = T(0);
    for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
      T tmp = abs(in[i]);
      if (tmp > local_max_data) {
        local_max_data = tmp;
      }
    }
    shared_max_data[tid] = local_max_data;
  } else {
    if (bid < n) {
      shared_max_data[tid] = abs(in[bid]);
    } else {
      shared_max_data[tid] = T(0);
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
struct FindAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx, const T* in,
                  const int num, T* out) {
    int block = 1024;
    int grid = (block - 1 + num) / block;
    grid = (grid > block) ? block : grid;

    framework::Tensor max;
    T* max_data = max.mutable_data<T>(phi::make_ddim({grid}), ctx.GetPlace());
    FindAbsMaxKernel<T><<<grid, block, 1024 * sizeof(T), ctx.stream()>>>(
        in, num, max_data);
    FindAbsMaxKernel<T><<<1, block, 1024 * sizeof(T), ctx.stream()>>>(
        max_data, grid, out);
  }
};

template struct FindAbsMaxFunctor<platform::CUDADeviceContext, float>;
template struct FindAbsMaxFunctor<platform::CUDADeviceContext,
                                  paddle::platform::float16>;

template <typename T>
__global__ void FindChannelAbsMaxKernelQuantAxis0(const T* in, const int n,
                                                  const int c, T* out) {
  int tid = threadIdx.x;
  int channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  extern __shared__ T shared_max_data[];
  shared_max_data[tid] = T(0);
  for (int i = tid; i < channel_size; i += blockDim.x) {
    T tmp = fabs(in_c[i]);
    if (tmp > shared_max_data[tid]) {
      shared_max_data[tid] = tmp;
    }
  }
  __syncthreads();
  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
__global__ void FindChannelAbsMaxKernelQuantAxis1(const T* in, const int n,
                                                  const int cin, const int cout,
                                                  T* out) {
  extern __shared__ T shared_max_data[];
  int cout_wh_size = n / cin;
  int wh_size = n / (cin * cout);

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  const T* in_current = in + tid * cout_wh_size + bid * wh_size;
  T local_max_data = T(0);
  for (int i = 0; i < wh_size; i++) {
    T tmp = fabs(in_current[i]);
    if (tmp > local_max_data) {
      local_max_data = tmp;
    }
  }
  shared_max_data[tid] = local_max_data;
  __syncthreads();

  int len = blockDim.x;
  for (int i = (len + 1) / 2; i > 0; len = i, i = (i + 1) / 2) {
    if (tid < i && tid + i < len &&
        shared_max_data[tid] < shared_max_data[tid + i]) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    if (i == 1) {
      i = 0;  // break the loop
    }
    __syncthreads();
  }
  if (tid == 0 && shared_max_data[0] > out[bid]) {
    out[bid] = shared_max_data[0];
  }
}

template <typename T>
struct FindChannelAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in_tensor, const int quant_axis,
                  T* out_abs_max) {
    PADDLE_ENFORCE_EQ(
        quant_axis == 0 || quant_axis == 1, true,
        platform::errors::InvalidArgument("'quant_axis' should be 0 or 1, but "
                                          "the received is %d",
                                          quant_axis));
    const int num = in_tensor.numel();
    auto in_dims = in_tensor.dims();
    const T* in_data = in_tensor.data<T>();
    if (quant_axis == 0) {
      int cout = in_dims[0];
      int grid = cout;
      int block = 1024;
      FindChannelAbsMaxKernelQuantAxis0<
          T><<<grid, block, block * sizeof(T), ctx.stream()>>>(
          in_data, num, cout, out_abs_max);
    } else if (quant_axis == 1) {
      int cin = in_dims[0];
      int cout = in_dims[1];
      int grid = cout;
      int max_threads = 1024;

#ifdef PADDLE_WITH_HIP
      hipMemset(out_abs_max, 0, sizeof(T) * cout);
#else
      hipMemset(out_abs_max, 0, sizeof(T) * cout);
#endif

      for (int i = 0; i < cin / max_threads; i++) {
        int block = max_threads;
        FindChannelAbsMaxKernelQuantAxis1<
            T><<<grid, block, block * sizeof(T), ctx.stream()>>>(
            in_data, num, cin, cout, out_abs_max);
        in_data += num / cin;
      }

      int block = cin % max_threads;
      if (block > 0) {
        FindChannelAbsMaxKernelQuantAxis1<
            T><<<grid, block, block * sizeof(T), ctx.stream()>>>(
            in_data, num, in_dims[0], in_dims[1], out_abs_max);
      }
    }
  }
};

template struct FindChannelAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
__global__ void ClipAndQuantKernel(const T* in, const T* scale,
                                   const int bin_cnt, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  T s = scale[0];
  T inv_s = inverse(s);
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    T x = in[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out[i] = round(v);
  }
}

template <typename T>
struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = (block - 1 + num) / block;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ClipAndQuantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, out_data);
  }
};

template struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, float>;

// ChannelClipAndQuantKernel for quant_axis is 0
template <typename T>
__global__ void ChannelClipAndQuantKernelQuantAxis0(const T* in, const T* scale,
                                                    const int bin_cnt,
                                                    const int64_t n,
                                                    const int c, T* out) {
  int tid = threadIdx.x;

  int64_t channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  T* out_c = out + blockIdx.x * channel_size;

  T s = scale[blockIdx.x];
  T inv_s = inverse(s);

  for (int64_t i = tid; i < channel_size; i += blockDim.x) {
    T x = in_c[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out_c[i] = round(v);
  }
}

// ChannelClipAndQuantKernel for quant_axis is N
template <typename T>
__global__ void ChannelClipAndQuantKernelQuantAxisN(
    const T* in, const T* scale, const int bin_cnt, const int64_t n,
    const int nScale, const int quant_stride, T* out) {
  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int64_t i = idx; i < n; i += blockDim.x * gridDim.x) {
    T s = scale[(i / quant_stride) % nScale];
    T inv_s = inverse(s);
    T x = in[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out[i] = round(v);
  }
}

template <typename T>
struct ChannelClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, const int quant_axis,
                  framework::Tensor* out) {
    PADDLE_ENFORCE_EQ(
        quant_axis == 0 || quant_axis == 1, true,
        platform::errors::InvalidArgument("'quant_axis' should be 0 or 1, but "
                                          "the received is %d",
                                          quant_axis));

    int64_t num = in.numel();
    auto in_dims = in.dims();
    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    if (quant_axis == 0) {
      int grid = in_dims[0];
      int block = 1024;
      ChannelClipAndQuantKernelQuantAxis0<T><<<grid, block, 0, ctx.stream()>>>(
          in_data, scale_data, bin_cnt, num, in_dims[0], out_data);
    } else {
      int quant_stride = 1;
      for (int i = quant_axis + 1; i < in_dims.size(); i++) {
        quant_stride *= in_dims[i];
      }
      int64_t block_size =
          std::min(num, static_cast<int64_t>(ctx.GetMaxThreadsPerBlock() / 4));
      int64_t max_threads =
          ctx.GetMaxPhysicalThreadCount();  // SM * block_per_SM
      const int64_t max_blocks = std::max(((max_threads - 1) / block_size + 1),
                                          static_cast<int64_t>(1));

      const int64_t grid_size =
          std::min(max_blocks, (num + block_size - 1) / block_size);

      ChannelClipAndQuantKernelQuantAxisN<T><<<grid_size, block_size>>>(
          in_data, scale_data, bin_cnt, num, in_dims[quant_axis], quant_stride,
          out_data);
    }
  }
};

template struct ChannelClipAndFakeQuantFunctor<platform::CUDADeviceContext,
                                               float>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(dequantize_linear,
                        ops::DeQuantizeLinearKernel<CUDA, float, float>,
                        ops::DeQuantizeLinearKernel<CUDA, int8_t, float>,
                        ops::DeQuantizeLinearKernel<CUDA, double, double>);

REGISTER_OP_CUDA_KERNEL(quantize_linear,
                        ops::QuantizeLinearKernel<CUDA, float>);
