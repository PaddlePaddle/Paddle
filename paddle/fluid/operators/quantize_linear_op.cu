#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/fake_dequantize_op.cu.h"
#include "paddle/fluid/operators/fake_quantize_op.cu.h"
#include "paddle/fluid/operators/quantize_linear_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
struct ChannelDequantizeFunctorV2<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& dev_ctx,
                  const framework::Tensor* in,
                  const framework::Tensor* scale,
                  T max_range,
                  const int quant_axis,
                  framework::Tensor* out) {
    auto in_dims = in->dims();
    const T* in_data = in->data<T>();
    T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());
    int64_t num = in->numel();
    const T* scale_factor = scale->data<T>();
    int64_t block_size = std::min(
        num, static_cast<int64_t>(dev_ctx.GetMaxThreadsPerBlock() / 4));
    int64_t max_threads =
        dev_ctx.GetMaxPhysicalThreadCount();  // SM * block_per_SM
    const int64_t max_blocks =
        std::max(((max_threads - 1) / block_size + 1), static_cast<int64_t>(1));
    const int64_t grid_size =
        std::min(max_blocks, (num + block_size - 1) / block_size);

    int quant_stride = 1;
    for (int i = quant_axis + 1; i < in_dims.size(); i++) {
      quant_stride *= in_dims[i];
    }

    DequantizeOneScaleQuantAxisN<T>
        <<<grid_size, block_size, 0, dev_ctx.stream()>>>(in_data,
                                                         scale_factor,
                                                         max_range,
                                                         num,
                                                         in_dims[quant_axis],
                                                         quant_stride,
                                                         out_data);
  }
};

template struct ChannelDequantizeFunctorV2<phi::GPUContext, float>;
template struct ChannelDequantizeFunctorV2<phi::GPUContext, double>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = phi::GPUContext;
REGISTER_OP_CUDA_KERNEL(dequantize_linear,
                        ops::DeQuantizeLinearKernel<CUDA, float, float>,
                        ops::DeQuantizeLinearKernel<CUDA, int8_t, float>,
                        ops::DeQuantizeLinearKernel<CUDA, double, double>);

REGISTER_OP_CUDA_KERNEL(quantize_linear,
                        ops::QuantizeLinearKernel<CUDA, float>);
