#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/platform/device_context.h>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/partial_sum_op.h"
#include "paddle/fluid/platform/float16.h"

namespace plat = paddle::platform;

namespace paddle {
namespace operators {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

using LoDTensor = framework::LoDTensor;
using Tensor = framework::Tensor;

template <class T>
__global__ void SumArrayPartialCUDAKernel(T **in, T *out, int64_t lod_length,
                                          size_t in_size, int64_t start_index,
                                          int64_t length, int64_t row_length) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < lod_length) {
    T total = static_cast<T>(0);
    int b_id = id / length;
    int b_offset = id % length;

    for (int i = 0; i < in_size; ++i) {
      const T *tmp = in[i];
      if (tmp) {
        total += tmp[start_index + b_id * row_length + b_offset];
      }
    }
    out[id] = total;
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void PartialSumGradCUDAKernel(T **res_grad, const T *out_grad,
                                         int64_t lod_length, size_t in_size,
                                         int64_t start_index, int64_t length,
                                         int64_t row_length) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < lod_length) {
    T total = static_cast<T>(0);
    int b_id = id / length;
    int b_offset = id % length;

    for (int i = 0; i < in_size; ++i) {
      T *tmp = res_grad[i];
      tmp[start_index + b_id * row_length + b_offset] = out_grad[i];
    }
    id += blockDim.x * gridDim.x;
  }
}

template <typename T>
class PartialSumOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto in_vars = ctx.MultiInput<Tensor>("X");
    Tensor *out = ctx.Output<Tensor>("Out");

    PADDLE_ENFORCE_EQ(
        in_vars[0] != nullptr, true,
        platform::errors::InvalidArgument("The input should not be null."));

    auto place = ctx.GetPlace();  // GPUPlace only now
    auto start_index = ctx.Attr<int>("start_index");
    auto length = ctx.Attr<int>("length");
    auto batch_size = in_vars[0]->dims()[0];
    if (length == -1) {
      length = in_vars[0]->dims()[1] - start_index;
    }

    constexpr size_t theory_sm_threads = 1024;
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    auto sm_count = max_threads / theory_sm_threads;
    size_t tile_size = 0;
    dim3 grids;
    dim3 blocks;
    auto ComputeKernelParameter = [&](size_t length) {
      if (length >= max_threads)
        tile_size = 1024;
      else if (length < max_threads && length > sm_count * 128)
        tile_size = 512;
      else if (length <= sm_count * 128)
        tile_size = 256;
      grids = dim3(CEIL_DIV(length, tile_size), 1, 1);
      blocks = dim3(tile_size, 1, 1);
    };

    auto lod_length = length * batch_size;
    auto row_length = in_vars[0]->dims()[1];
    auto in_num = in_vars.size();

    std::vector<const T *> in_data;
    for (int i = 0; i < in_num; ++i) {
      in_data.emplace_back(in_vars[i]->data<T>());
    }

    if (!in_data.empty()) {
      auto tmp_in_array = memory::Alloc(dev_ctx, in_data.size() * sizeof(T *));

      memory::Copy(BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                   tmp_in_array->ptr(), platform::CPUPlace(),
                   reinterpret_cast<void *>(in_data.data()),
                   in_data.size() * sizeof(T *), dev_ctx.stream());

      T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());
      ComputeKernelParameter(lod_length);
      SumArrayPartialCUDAKernel<T><<<grids, blocks, 0, stream>>>(
          in_array_data, out->data<T>(), lod_length, in_data.size(),
          start_index, length, row_length);
    }
  }
};

template <typename T>
class PartialSumGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const Tensor *out_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto ins = ctx.MultiInput<LoDTensor>("X");
    auto outs = ctx.MultiOutput<LoDTensor>(framework::GradVarName("X"));

    PADDLE_ENFORCE_EQ(
        ins[0] != nullptr, true,
        platform::errors::InvalidArgument("The input should not be null."));
    auto start_index = ctx.Attr<int>("start_index");
    auto length = ctx.Attr<int>("length");
    if (length == -1) {
      length = ins[0]->dims()[1] - start_index;
    }

    // initialize
    auto &place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();
    for (size_t i = 0; i < outs.size(); ++i) {
      outs[i]->mutable_data<T>(ctx.GetPlace());
      auto dxt = framework::EigenVector<T>::Flatten(*outs[i]);
      dxt.device(place) = dxt.constant(static_cast<T>(0));
    }

    auto batch_size = ins[0]->dims()[0];
    if (length == -1) {
      length = ins[0]->dims()[1] - start_index;
    }
    auto lod_length = length * batch_size;
    auto row_length = ins[0]->dims()[1];
    auto out_num = outs.size();

    constexpr size_t theory_sm_threads = 1024;
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    auto sm_count = max_threads / theory_sm_threads;
    size_t tile_size = 0;
    dim3 grids;
    dim3 blocks;
    auto ComputeKernelParameter = [&](size_t length) {
      if (length >= max_threads)
        tile_size = 1024;
      else if (length < max_threads && length > sm_count * 128)
        tile_size = 512;
      else if (length <= sm_count * 128)
        tile_size = 256;
      grids = dim3(CEIL_DIV(length, tile_size), 1, 1);
      blocks = dim3(tile_size, 1, 1);
    };

    std::vector<const T *> out_data;
    for (int i = 0; i < out_num; ++i) {
      out_data.emplace_back(outs[i]->data<T>());
    }

    if (!out_data.empty()) {
      auto tmp_out_array =
          memory::Alloc(dev_ctx, out_data.size() * sizeof(T *));

      memory::Copy(BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                   tmp_out_array->ptr(), platform::CPUPlace(),
                   reinterpret_cast<void *>(out_data.data()),
                   out_data.size() * sizeof(T *), dev_ctx.stream());

      T **out_grad_data = reinterpret_cast<T **>(tmp_out_array->ptr());
      ComputeKernelParameter(lod_length);
      PartialSumGradCUDAKernel<T><<<grids, blocks, 0, stream>>>(
          out_grad_data, out_grad->data<T>(), lod_length, out_data.size(),
          start_index, length, row_length);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(partial_sum, ops::PartialSumOpCUDAKernel<float>,
                        ops::PartialSumOpCUDAKernel<double>,
                        ops::PartialSumOpCUDAKernel<int>,
                        ops::PartialSumOpCUDAKernel<int64_t>,
                        ops::PartialSumOpCUDAKernel<plat::float16>);

REGISTER_OP_CUDA_KERNEL(partial_sum_grad,
                        ops::PartialSumGradOpCUDAKernel<float>,
                        ops::PartialSumGradOpCUDAKernel<double>,
                        ops::PartialSumGradOpCUDAKernel<int>,
                        ops::PartialSumGradOpCUDAKernel<int64_t>,
                        ops::PartialSumGradOpCUDAKernel<plat::float16>);
