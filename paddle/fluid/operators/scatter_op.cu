/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/gather_op.h"
#include "paddle/fluid/operators/scatter.cu.h"
#include "paddle/fluid/operators/scatter_op.h"

namespace paddle {
namespace operators {

template <typename T>
class ScatterOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *X = ctx.Input<Tensor>("X");
    auto *Ids = ctx.Input<Tensor>("Ids");
    auto *Updates = ctx.Input<Tensor>("Updates");
    auto *Out = ctx.Output<Tensor>("Out");
    bool overwrite = ctx.Attr<bool>("overwrite");

    Out->ShareDataWith(*X);
    GPUScatterAssign<T>(ctx, *Updates, *Ids, Out, overwrite);
  }
};

template <typename T>
class ScatterGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *dX = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *dUpdates = ctx.Output<Tensor>(framework::GradVarName("Updates"));
    auto *Ids = ctx.Input<Tensor>("Ids");
    auto *dOut = ctx.Input<Tensor>(framework::GradVarName("Out"));
    if (dX) {
      // In place gradient: dX = dO
      framework::TensorCopy(*dOut, ctx.GetPlace(), dX);
    }
    if (dUpdates) {
      dUpdates->mutable_data<T>(ctx.GetPlace());
      // Gradient by Gather: dUpdates = dO[Ids]
      GPUGather<T>(ctx.device_context(), *dOut, *Ids, dUpdates);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(scatter, ops::ScatterOpCUDAKernel<float>);
REGISTER_OP_CUDA_KERNEL(scatter_grad, ops::ScatterGradOpCUDAKernel<float>);
