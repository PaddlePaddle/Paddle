#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

/* Acknowledgement: the following code is strongly inspired by
https://github.com/caffe2/caffe2/blob/master/caffe2/operators/lstm_unit_op_gpu.cu
*/

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/cross_entropy_op.h"
#include "paddle/fluid/operators/lstm_unit_op.h"
#include "paddle/fluid/platform/assert.h"
#include "paddle/fluid/platform/hostdevice.h"

namespace paddle {
namespace operators {

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename Dtype>
__device__ Dtype cuda_sigmoid(const Dtype x) {
  return Dtype(1) / (Dtype(1) + exp(-x));
}

template <typename Dtype>
__device__ Dtype cuda_tanh(const Dtype x) {
  return Dtype(1 - exp(-2. * x)) / (Dtype(1) + exp(-2. * x));
}

template <typename T>
__global__ void LSTMUnitKernel(const int nthreads, const int dim,
                               const T* C_prev, const T* X, T* C, T* H,
                               const T forget_bias) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int n = index / dim;
    const int d = index % dim;

    const T* X_offset = X + 4 * dim * n;
    const T i = cuda_sigmoid(X_offset[d]);
    const T f = cuda_sigmoid(X_offset[1 * dim + d] + forget_bias);
    const T o = cuda_sigmoid(X_offset[2 * dim + d]);
    const T g = cuda_tanh(X_offset[3 * dim + d]);
    const T c_prev = C_prev[index];
    const T c = f * c_prev + i * g;
    C[index] = c;
    const T tanh_c = cuda_tanh(c);
    H[index] = o * tanh_c;
  }
}

template <typename T>
__global__ void LSTMUnitGradientKernel(const int nthreads, const int dim,
                                       const T* C_prev, const T* X, const T* C,
                                       const T* H, const T* C_diff,
                                       const T* H_diff, T* C_prev_diff,
                                       T* X_diff, const T forget_bias) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int n = index / dim;
    const int d = index % dim;
    const T* X_offset = X + 4 * dim * n;
    T* c_prev_diff = C_prev_diff + index;
    T* X_diff_offset = X_diff + 4 * dim * n;
    T* i_diff = X_diff_offset + d;
    T* f_diff = X_diff_offset + 1 * dim + d;
    T* o_diff = X_diff_offset + 2 * dim + d;
    T* g_diff = X_diff_offset + 3 * dim + d;

    const T i = cuda_sigmoid(X_offset[d]);
    const T f = cuda_sigmoid(X_offset[1 * dim + d] + forget_bias);
    const T o = cuda_sigmoid(X_offset[2 * dim + d]);
    const T g = cuda_tanh(X_offset[3 * dim + d]);
    const T c_prev = C_prev[index];
    const T c = C[index];
    const T tanh_c = cuda_tanh(c);
    const T c_term_diff =
        C_diff[index] + H_diff[index] * o * (1 - tanh_c * tanh_c);
    *c_prev_diff = c_term_diff * f;
    *i_diff = c_term_diff * g * i * (1 - i);
    *f_diff = c_term_diff * c_prev * f * (1 - f);
    *o_diff = H_diff[index] * tanh_c * o * (1 - o);
    *g_diff = c_term_diff * i * (1 - g * g);
  }
}

template <typename T>
class LstmUnitOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");

    auto* x_tensor = ctx.Input<framework::Tensor>("X");
    auto* c_prev_tensor = ctx.Input<framework::Tensor>("C_prev");
    auto* c_tensor = ctx.Output<framework::Tensor>("C");
    auto* h_tensor = ctx.Output<framework::Tensor>("H");

    auto forget_bias = static_cast<T>(ctx.Attr<float>("forget_bias"));

    int b_size = c_tensor->dims()[0];
    int D = c_tensor->dims()[1];

    const T* X = x_tensor->data<T>();
    const T* C_prev = c_prev_tensor->data<T>();

    T* C = c_tensor->mutable_data<T>(ctx.GetPlace());
    T* H = h_tensor->mutable_data<T>(ctx.GetPlace());

    int block = 512;
    int n = b_size * D;
    int grid = (n + block - 1) / block;

    LSTMUnitKernel<T><<<grid, block>>>(n, D, C_prev, X, C, H, forget_bias);
  }
};

template <typename T>
class LstmUnitGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");

    auto x_tensor = ctx.Input<Tensor>("X");
    auto c_prev_tensor = ctx.Input<Tensor>("C_prev");
    auto c_tensor = ctx.Input<Tensor>("C");
    auto h_tensor = ctx.Input<Tensor>("H");

    auto hdiff_tensor = ctx.Input<Tensor>(framework::GradVarName("H"));
    auto cdiff_tensor = ctx.Input<Tensor>(framework::GradVarName("C"));

    auto xdiff_tensor = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto c_prev_diff_tensor =
        ctx.Output<Tensor>(framework::GradVarName("C_prev"));

    auto* X = x_tensor->data<T>();
    auto* C_prev = c_prev_tensor->data<T>();
    auto* C = c_tensor->data<T>();
    auto* H = h_tensor->data<T>();

    auto* H_diff = hdiff_tensor->data<T>();
    auto* C_diff = cdiff_tensor->data<T>();

    auto* C_prev_diff = c_prev_diff_tensor->mutable_data<T>(ctx.GetPlace());
    auto* X_diff = xdiff_tensor->mutable_data<T>(ctx.GetPlace());

    int N = c_tensor->dims()[0];
    int D = c_tensor->dims()[1];

    auto forget_bias = static_cast<T>(ctx.Attr<float>("forget_bias"));

    int block = 512;
    int n = N * D;
    int grid = (n + block - 1) / block;

    LSTMUnitGradientKernel<T><<<grid, block>>>(n, D, C_prev, X, C, H, C_diff,
                                               H_diff, C_prev_diff, X_diff,
                                               forget_bias);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(lstm_unit, ops::LstmUnitOpCUDAKernel<float>,
                        ops::LstmUnitOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(lstm_unit_grad, ops::LstmUnitGradOpCUDAKernel<float>,
                        ops::LstmUnitGradOpCUDAKernel<double>);
