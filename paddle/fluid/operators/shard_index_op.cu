#include "hip/hip_runtime.h"
//   Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/shard_index_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

template <typename InT, typename OutT>
__global__ void ShardIndexInner(const InT* p_in_data, OutT* p_out_data,
                                const int64_t numel, const int shard_range,
                                const int shard_id) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel) {
    if (p_in_data[idx] / shard_range == shard_id) {
      p_out_data[idx] = p_in_data[idx] % shard_range;
    } else {
      p_out_data[idx] = -1;
    }
  }
}

using LoDTensor = framework::LoDTensor;

template <typename T>
class ShardIndexCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");
    int shard_range = context.Attr<int>("shard_range");
    int shard_id = context.Attr<int>("shard_id");
    PADDLE_ENFORCE_GE(shard_id, 0);
    PADDLE_ENFORCE_GT(shard_range, 0);

    out->Resize(in->dims());
    out->set_lod(in->lod());
    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    ShardIndexInner<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                          PADDLE_CUDA_NUM_THREADS,
                      PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_data, out_data, numel, shard_range, shard_id);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(shard_index, ops::ShardIndexCUDAKernel<int>,
                        ops::ShardIndexCUDAKernel<int64_t>);
