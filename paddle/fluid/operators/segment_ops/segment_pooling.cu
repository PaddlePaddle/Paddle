#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/segment_ops/segment_pooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_param_config.h"
#include "paddle/fluid/platform/macros.h"

namespace paddle {
namespace operators {

template <typename T>
DEVICE inline void max_functor(const T& x, T* y) {
  *y = *y > x ? *y : x;
}
template <typename T>
DEVICE inline void min_functor(const T& x, T* y) {
  *y = *y < x ? *y : x;
}

template <typename T, typename Index, int OuterDimTileSize>
__global__ void SortedSegmentSumCustomKernel(const Index input_outer_dim_size,
                                             const Index inner_dim_size,
                                             const Index output_outer_dim_size,
                                             const Index* segment_ids,
                                             const T* input, T* output,
                                             const Index total_stripe_count) {
  CUDA_KERNEL_LOOP(stripe_index, total_stripe_count) {
    const Index segment_offset = stripe_index % inner_dim_size;
    const Index input_outer_dim_index_base =
        stripe_index / inner_dim_size * Index(OuterDimTileSize);

    T sum = T(0);
    Index first_segment_id = segment_ids[input_outer_dim_index_base];
    Index last_output_segment_id = output_outer_dim_size;

    const Index actual_stripe_height =
        min(Index(OuterDimTileSize),
            input_outer_dim_size - input_outer_dim_index_base);
    for (Index j = 0; j < actual_stripe_height; j++) {
      Index current_output_segment_id =
          segment_ids[input_outer_dim_index_base + j];
      // Decide whether to write result to global memory.
      // Result is only written to global memory if we move
      // to another segment. Otherwise we can keep accumulating
      // locally.
      if (current_output_segment_id > last_output_segment_id) {
        const Index output_index =
            last_output_segment_id * inner_dim_size + segment_offset;
        // decide whether to write result to global memory using atomic
        // operations
        if (last_output_segment_id == first_segment_id) {
          platform::CudaAtomicAdd(output + output_index, sum);
        } else {
          *(output + output_index) = sum;
        }
        sum = T(0);
      }
      sum += input[(input_outer_dim_index_base + j) * inner_dim_size +
                   segment_offset];
      // sum += __ldg(input + (input_outer_dim_index_base + j) * inner_dim_size
      // +segment_offset);
      last_output_segment_id = current_output_segment_id;
    }
    // For the last result in a strip, always write using atomic operations
    // due to possible race conditions with threads computing
    // the following strip.
    const Index output_index =
        last_output_segment_id * inner_dim_size + segment_offset;
    platform::CudaAtomicAdd(output + output_index, sum);
  }
}

template <typename T, typename Index, int OuterDimTileSize>
__global__ void SortedSegmentMaxCustomKernel(
    const Index input_outer_dim_size, const Index inner_dim_size,
    const Index output_outer_dim_size, const Index* segment_ids, const T* input,
    T* output, const Index total_stripe_count, const bool MAX = false) {
  CUDA_KERNEL_LOOP(stripe_index, total_stripe_count) {
    const Index segment_offset = stripe_index % inner_dim_size;
    const Index input_outer_dim_index_base =
        stripe_index / inner_dim_size * Index(OuterDimTileSize);

    T minmax = static_cast<T>(-FLT_MAX);
    Index first_segment_id = segment_ids[input_outer_dim_index_base];
    Index last_output_segment_id = output_outer_dim_size;

    const Index actual_stripe_height =
        min(Index(OuterDimTileSize),
            input_outer_dim_size - input_outer_dim_index_base);
    // -1 is for the start value when interval_id = 0
    Index previous_segment_id = -1;
    if (input_outer_dim_index_base > 0) {
      previous_segment_id = segment_ids[input_outer_dim_index_base - 1];
    }
    for (Index interval_id = previous_segment_id + 1;
         interval_id < first_segment_id; ++interval_id) {
      *(output + interval_id * inner_dim_size + segment_offset) = 0;
    }

    for (Index j = 0; j < actual_stripe_height; j++) {
      Index current_output_segment_id =
          segment_ids[input_outer_dim_index_base + j];

      if (current_output_segment_id > last_output_segment_id) {
        const Index output_index =
            last_output_segment_id * inner_dim_size + segment_offset;
        if (last_output_segment_id == first_segment_id) {
          platform::CudaAtomicMax(output + output_index, minmax);
        } else {
          *(output + output_index) = minmax;
        }
        // reset the interval value which do not have corresponding ids.
        for (Index interval_index = 1;
             interval_index <
             current_output_segment_id - last_output_segment_id;
             ++interval_index) {
          *(output + output_index + interval_index * inner_dim_size) = 0;
        }
        minmax = static_cast<T>(-FLT_MAX);
      }
      max_functor<T>(input[(input_outer_dim_index_base + j) * inner_dim_size +
                           segment_offset],
                     &minmax);
      last_output_segment_id = current_output_segment_id;
    }
    const Index output_index =
        last_output_segment_id * inner_dim_size + segment_offset;
    platform::CudaAtomicMax(output + output_index, minmax);
  }
}

template <typename T, typename Index, int OuterDimTileSize>
__global__ void SortedSegmentMinCustomKernel(
    const Index input_outer_dim_size, const Index inner_dim_size,
    const Index output_outer_dim_size, const Index* segment_ids, const T* input,
    T* output, const Index total_stripe_count, const bool MAX = false) {
  CUDA_KERNEL_LOOP(stripe_index, total_stripe_count) {
    const Index segment_offset = stripe_index % inner_dim_size;
    const Index input_outer_dim_index_base =
        stripe_index / inner_dim_size * Index(OuterDimTileSize);

    T minmax = static_cast<T>(FLT_MAX);
    Index first_segment_id = segment_ids[input_outer_dim_index_base];
    Index last_output_segment_id = output_outer_dim_size;

    const Index actual_stripe_height =
        min(Index(OuterDimTileSize),
            input_outer_dim_size - input_outer_dim_index_base);
    // -1 is for the start value when interval_id = 0
    Index previous_segment_id = -1;
    if (input_outer_dim_index_base > 0) {
      previous_segment_id = segment_ids[input_outer_dim_index_base - 1];
    }
    for (Index interval_id = previous_segment_id + 1;
         interval_id < first_segment_id; ++interval_id) {
      *(output + interval_id * inner_dim_size + segment_offset) = 0;
    }

    for (Index j = 0; j < actual_stripe_height; j++) {
      Index current_output_segment_id =
          segment_ids[input_outer_dim_index_base + j];

      if (current_output_segment_id > last_output_segment_id) {
        const Index output_index =
            last_output_segment_id * inner_dim_size + segment_offset;
        if (last_output_segment_id == first_segment_id) {
          platform::CudaAtomicMin(output + output_index, minmax);
        } else {
          *(output + output_index) = minmax;
        }
        // reset the interval value which do not have corresponding ids.
        for (Index interval_index = 1;
             interval_index <
             current_output_segment_id - last_output_segment_id;
             ++interval_index) {
          *(output + output_index + interval_index * inner_dim_size) = 0;
        }
        minmax = static_cast<T>(FLT_MAX);
      }
      min_functor<T>(input[(input_outer_dim_index_base + j) * inner_dim_size +
                           segment_offset],
                     &minmax);
      last_output_segment_id = current_output_segment_id;
    }
    const Index output_index =
        last_output_segment_id * inner_dim_size + segment_offset;
    platform::CudaAtomicMin(output + output_index, minmax);
  }
}

template <typename T, typename Index, int OuterDimTileSize>
__global__ void SortedSegmentIndexGradKernel(const Index input_outer_dim_size,
                                             const Index inner_dim_size,
                                             const Index output_outer_dim_size,
                                             const Index* segment_ids,
                                             const T* input, const T* output,
                                             const T* out_grad, T* in_grad,
                                             const Index total_stripe_count) {
  CUDA_KERNEL_LOOP(stripe_index, total_stripe_count) {
    const Index segment_offset = stripe_index % inner_dim_size;
    const Index input_outer_dim_index_base =
        stripe_index / inner_dim_size * Index(OuterDimTileSize);

    const Index actual_stripe_height =
        min(Index(OuterDimTileSize),
            input_outer_dim_size - input_outer_dim_index_base);
    for (Index j = 0; j < actual_stripe_height; j++) {
      Index current_output_segment_id =
          segment_ids[input_outer_dim_index_base + j];
      Index input_index =
          (input_outer_dim_index_base + j) * inner_dim_size + segment_offset;
      Index output_index =
          current_output_segment_id * inner_dim_size + segment_offset;
      if (input[input_index] == output[output_index]) {
        in_grad[input_index] = out_grad[output_index];
      }
    }
  }
}

template <typename T, typename Index>
void SegmentPoolCUDAFunctor(const platform::CUDADeviceContext& ctx,
                            const framework::Tensor& input,
                            const framework::Tensor& segment_ids,
                            framework::Tensor* output,
                            const std::string pooltype = "SUM") {
  const Index input_total_size = input.numel();
  const Index input_outer_dim_size = segment_ids.dims()[0];
  const Index output_rows = output->dims()[0];
  const Index input_inner_dim_size = input_total_size / input_outer_dim_size;

  const Index OuterDimTileSize = 8;
  const Index input_outer_dim_num_stripe =
      (input_outer_dim_size + OuterDimTileSize - 1) / OuterDimTileSize;

  const Index total_stripe_count =
      input_inner_dim_size * input_outer_dim_num_stripe;

  auto config = platform::GetGpuLaunchConfig1D(ctx, total_stripe_count);

  if (pooltype == "SUM") {
    SortedSegmentSumCustomKernel<T, Index, OuterDimTileSize><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, ctx.stream()>>>(
        input_outer_dim_size, input_inner_dim_size, output_rows,
        segment_ids.data<Index>(), input.data<T>(), output->data<T>(),
        total_stripe_count);
  } else if (pooltype == "MAX") {
    SortedSegmentMaxCustomKernel<T, Index, OuterDimTileSize><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, ctx.stream()>>>(
        input_outer_dim_size, input_inner_dim_size, output_rows,
        segment_ids.data<Index>(), input.data<T>(), output->data<T>(),
        total_stripe_count);
  } else if (pooltype == "MIN") {
    SortedSegmentMinCustomKernel<T, Index, OuterDimTileSize><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, ctx.stream()>>>(
        input_outer_dim_size, input_inner_dim_size, output_rows,
        segment_ids.data<Index>(), input.data<T>(), output->data<T>(),
        total_stripe_count);
  } else {
    PADDLE_THROW(platform::errors::Unimplemented("Not support yet."));
  }
}

template <typename T, typename Index>
void SegmentPoolCUDAGradFunctor(const platform::CUDADeviceContext& ctx,
                                const framework::Tensor& input,
                                const framework::Tensor& segment_ids,
                                const framework::Tensor& output,
                                const framework::Tensor& out_grad,
                                framework::Tensor* in_grad,
                                const std::string pooltype = "SUM") {
  const Index input_total_size = input.numel();
  const Index input_outer_dim_size = segment_ids.dims()[0];
  const Index output_rows = output.dims()[0];
  const Index input_inner_dim_size = input_total_size / input_outer_dim_size;

  const Index OuterDimTileSize = 8;
  const Index input_outer_dim_num_stripe =
      (input_outer_dim_size + OuterDimTileSize - 1) / OuterDimTileSize;

  const Index total_stripe_count =
      input_inner_dim_size * input_outer_dim_num_stripe;

  auto config = platform::GetGpuLaunchConfig1D(ctx, total_stripe_count);

  if (pooltype == "MAX" || pooltype == "MIN") {
    SortedSegmentIndexGradKernel<T, Index, OuterDimTileSize><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, ctx.stream()>>>(
        input_outer_dim_size, input_inner_dim_size, output_rows,
        segment_ids.data<Index>(), input.data<T>(), output.data<T>(),
        out_grad.data<T>(), in_grad->data<T>(), total_stripe_count);
  } else {
    PADDLE_THROW(platform::errors::Unimplemented("Not support yet."));
  }
}

template <typename T, typename IndexT>
class SegmentPoolFunctor<platform::CUDADeviceContext, T, IndexT> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& segments, framework::Tensor* output,
                  framework::Tensor* index,
                  const std::string pooltype = "SUM") {
    if (pooltype == "MEAN") {
      PADDLE_THROW(platform::errors::Unimplemented("Not support yet."));
    } else if (pooltype == "SUM") {
      SegmentPoolCUDAFunctor<T, IndexT>(context, input, segments, output,
                                        pooltype);
    } else if (pooltype == "MAX") {
      SegmentPoolCUDAFunctor<T, IndexT>(context, input, segments, output,
                                        pooltype);
    } else if (pooltype == "MIN") {
      SegmentPoolCUDAFunctor<T, IndexT>(context, input, segments, output,
                                        pooltype);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported segment pooling operation, Only MEAN, SUM, MAX, MIN "
          "available, but got %s.",
          pooltype));
    }
  }
};

template <typename T, typename IndexT>
class SegmentPoolGradFunctor<platform::CUDADeviceContext, T, IndexT> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& out_grad,
                  const framework::Tensor& segments, framework::Tensor* in_grad,
                  const framework::Tensor* index = nullptr,
                  const std::string pooltype = "SUM") {
    if (pooltype == "MAX" || pooltype == "MIN") {
      SegmentPoolCUDAGradFunctor<T, IndexT>(context, input, segments, output,
                                            out_grad, in_grad, pooltype);
      return;
    }

    if (pooltype == "MEAN") {
      PADDLE_THROW(platform::errors::Unimplemented("Not support yet."));
    } else if (pooltype == "SUM") {
      GPUGather<T, IndexT>(context, out_grad, segments, in_grad);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported segment pooling operation, Only MEAN, SUM, MAX, MIN "
          "available, but got %s.",
          pooltype));
    }
  }
};

using CUDA = paddle::platform::CUDADeviceContext;
template class SegmentPoolFunctor<CUDA, float, int>;
template class SegmentPoolFunctor<CUDA, float, int64_t>;
template class SegmentPoolFunctor<CUDA, double, int>;
template class SegmentPoolFunctor<CUDA, double, int64_t>;
template class SegmentPoolGradFunctor<CUDA, float, int>;
template class SegmentPoolGradFunctor<CUDA, float, int64_t>;
template class SegmentPoolGradFunctor<CUDA, double, int>;
template class SegmentPoolGradFunctor<CUDA, double, int64_t>;

}  // namespace operators
}  // namespace paddle
