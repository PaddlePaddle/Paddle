#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/linspace_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void LinspaceKernel(T start, T step, int64_t size, T* out) {
  CUDA_1D_KERNEL_LOOP(index, size) { out[index] = start + step * index; }
}

template <typename T>
__global__ void LinspaceSpecialKernel(T start, T* out) {
  out[0] = start;
}

template <typename T>
class CUDALinspaceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* start_t = context.Input<framework::Tensor>("Start");
    auto* stop_t = context.Input<framework::Tensor>("Stop");
    auto* num_t = context.Input<framework::Tensor>("Num");
    auto* out = context.Output<framework::Tensor>("Out");

    framework::Tensor n;
    framework::TensorCopy(*start_t, platform::CPUPlace(), &n);
    T start = n.data<T>()[0];
    framework::TensorCopy(*stop_t, platform::CPUPlace(), &n);
    T stop = n.data<T>()[0];
    framework::TensorCopy(*num_t, platform::CPUPlace(), &n);
    int32_t num = n.data<int32_t>()[0];

    PADDLE_ENFORCE(num > 0, "The num of linspace op should be larger than 0.");

    out->Resize(framework::make_ddim({num}));
    T* out_data = out->mutable_data<T>(context.GetPlace());

    T step = 0;
    if (num != 1) {
      step = (stop - start) / (num - 1);
    }

    auto stream = context.cuda_device_context().stream();
    int block = 512;
    int grid = (num + block - 1) / block;
    LinspaceKernel<T><<<grid, block, 0, stream>>>(start, step, num, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(linspace, ops::CUDALinspaceKernel<float>,
                        ops::CUDALinspaceKernel<double>);
