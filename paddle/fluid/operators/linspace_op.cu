#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/data_type_transform.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/linspace_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void LinspaceKernel(T start, T stop, double step, int64_t size,
                               T* out) {
  int64_t index = blockIdx.x * blockDim.x + threadIdx.x;

  for (; index < size; index += blockDim.x * gridDim.x) {
    if (index < size / 2) {
      out[index] = static_cast<T>(start + step * index);
    } else {
      out[index] = static_cast<T>(stop - step * (size - index - 1));
    }
  }
}

template <typename T>
__global__ void LinspaceSpecialKernel(T start, T* out) {
  out[0] = static_cast<T>(start);
}

template <typename T>
class CUDALinspaceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* pre_start = context.Input<framework::Tensor>("Start");
    auto* pre_stop = context.Input<framework::Tensor>("Stop");
    auto* num_t = context.Input<framework::Tensor>("Num");
    auto* out = context.Output<framework::Tensor>("Out");
    auto dtype = static_cast<framework::proto::VarType::Type>(
        context.Attr<int>("dtype"));

    Tensor start_t;
    Tensor stop_t;
    auto start_dtype =
        framework::OpKernelType(pre_start->type(), context.GetPlace());
    auto stop_dtype =
        framework::OpKernelType(pre_stop->type(), context.GetPlace());
    auto out_dtype = framework::OpKernelType(dtype, context.GetPlace());
    framework::TransDataType(start_dtype, out_dtype, *pre_start, &start_t);
    framework::TransDataType(stop_dtype, out_dtype, *pre_stop, &stop_t);

    framework::Tensor n_start;
    framework::Tensor n_stop;
    framework::Tensor n_num;
    framework::TensorCopy(start_t, platform::CPUPlace(), &n_start);
    T start = n_start.data<T>()[0];
    framework::TensorCopy(stop_t, platform::CPUPlace(), &n_stop);
    T stop = n_stop.data<T>()[0];
    framework::TensorCopy(*num_t, platform::CPUPlace(), &n_num);
    int64_t num = static_cast<int64_t>(n_num.data<int32_t>()[0]);

    PADDLE_ENFORCE_GT(num, 0, platform::errors::InvalidArgument(
                                  "The num of linspace op should be larger "
                                  "than 0, but received num is %d",
                                  num));

    out->Resize(framework::make_ddim({num}));
    T* out_data = out->mutable_data<T>(context.GetPlace());

    double step = 0;
    auto stream = context.cuda_device_context().stream();
    int block = 512;
    int grid = (num + block - 1) / block;
    if (num != 1) {
      step = (static_cast<double>(stop - start)) / (num - 1);
      LinspaceKernel<T><<<grid, block, 0, stream>>>(start, stop, step, num,
                                                    out_data);
    } else {
      LinspaceSpecialKernel<T><<<grid, block, 0, stream>>>(start, out_data);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(linspace, ops::CUDALinspaceKernel<float>,
                        ops::CUDALinspaceKernel<int32_t>,
                        ops::CUDALinspaceKernel<int64_t>,
                        ops::CUDALinspaceKernel<double>);
