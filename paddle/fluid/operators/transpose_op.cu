#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <limits>
#include <utility>

#include "paddle/fluid/framework/gpu_utils.h"
#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/gpu_launch_param_config.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using Dim3 = framework::Dim3;
using Index3 = framework::Index3;

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

struct EqualTo {
  constexpr bool operator()(int a, int b) const { return a == b; }
};

struct GreaterThan {
  constexpr bool operator()(int a, int b) const { return a > b; }
};

// Value can be decided in compile time.
template <typename FUN, int INT_32 = 32>
constexpr bool CheckProperTileSize(int tile_long, int tile_short, int size_T,
                                   FUN op) {
  return (size_T == 16 && ((tile_long == INT_32 && op(tile_short, 4)) ||
                           (tile_long == 2 * INT_32 && op(tile_short, 4)) ||
                           (tile_long == 4 * INT_32 && op(tile_short, 4)) ||
                           (tile_long == 8 * INT_32 && op(tile_short, 2)))) ||
         (size_T == 8 && ((tile_long == INT_32 && op(tile_short, 15)) ||
                          (tile_long == 2 * INT_32 && op(tile_short, 15)) ||
                          (tile_long == 4 * INT_32 && op(tile_short, 8)) ||
                          (tile_long == 8 * INT_32 && op(tile_short, 4)) ||
                          (tile_long == 16 * INT_32 && op(tile_short, 2)))) ||
         ((size_T == 4 || size_T == 2 || size_T == 1) &&
          ((tile_long == INT_32 && op(tile_short, 15)) ||
           (tile_long == 2 * INT_32 && op(tile_short, 15)) ||
           (tile_long == 4 * INT_32 && op(tile_short, 8)) ||
           (tile_long == 8 * INT_32 && op(tile_short, 4)) ||
           (tile_long == 16 * INT_32 && op(tile_short, 2)) ||
           (tile_long == 16 * INT_32 && op(tile_short, 2))));
}

constexpr bool CheckLongTileSize(int tile_long, int tile_short, int size_T) {
  return CheckProperTileSize(tile_long, tile_short, size_T, EqualTo());
}

constexpr bool CheckOutsideTileSize(int tile_long, int tile_short, int size_T) {
  return CheckProperTileSize(tile_long, tile_short, size_T, GreaterThan());
}

constexpr bool CheckNonLongTileSize(int tile_long, int tile_short, int size_T) {
  return !CheckOutsideTileSize(tile_long, tile_short, size_T) &&
         (CheckOutsideTileSize(tile_long * 2, tile_short, size_T) ||
          CheckOutsideTileSize(tile_long, tile_short + 1, size_T)) &&
         !CheckLongTileSize(tile_long, tile_short, size_T);
}

// Use SM to do data transfer, load a tile into SM then store out.
// All tile read and write are colascing, so can speedup memory copy
template <typename T, int NumThreads, int TileX, int TileY>
__global__ void TilingSwapDim1And2(const T* __restrict__ input, Dim3 input_dims,
                                   T* __restrict__ output) {
  assert(blockDim.x == NumThreads);
  assert(blockDim.y == 1);
  assert(blockDim.z == 1);
  assert(gridDim.y == 1);
  assert(gridDim.z == 1);

  constexpr int BlockReadRows = NumThreads / TileY;
  constexpr int BlockWriteRows = NumThreads / TileX;

  // One extra line in the inner dimension to avoid share memory bank conflict.
  __shared__ __align__(
      alignof(T)) char share_mem_ptr[TileX * (TileY + 1) * sizeof(T)];
  typedef T(*ShareMemory)[TileY + 1];

  ShareMemory tile_sm = reinterpret_cast<ShareMemory>(share_mem_ptr);

  int x = threadIdx.x;

  Dim3 output_dims = {
      input_dims[0], input_dims[2], input_dims[1],
  };

  // Align dim to Tiles
  Dim3 tile_aligned_input_dim = {
      input_dims[0], (input_dims[1] + TileX - 1) / TileX,
      (input_dims[2] + TileY - 1) / TileY,
  };

  // Converts block idx to tile index, each block process a tile
  Index3 input_block_tile_index =
      ConvertTensorIndex(blockIdx.x, tile_aligned_input_dim);

  // Compute real index align to tile:0, 32, 64...
  Index3 block_tile_index_in_input = {
      input_block_tile_index[0], input_block_tile_index[1] * TileX,
      input_block_tile_index[2] * TileY,
  };

  // Compute block flat index against input dims.
  int input_origin_block_flat_index =
      FlatTensorIndex(block_tile_index_in_input, input_dims);

  bool full_tile = true;
  int tile_width = TileY;

  // Last row is not full.
  if (input_block_tile_index[2] == tile_aligned_input_dim[2] - 1) {
    tile_width = input_dims[2] - (tile_aligned_input_dim[2] - 1) * TileY;
    full_tile &= false;
  }

  int tile_height = TileX;

  if (input_block_tile_index[1] == tile_aligned_input_dim[1] - 1) {
    tile_height = input_dims[1] - (tile_aligned_input_dim[1] - 1) * TileX;
    full_tile &= false;
  }

  constexpr int in_effective_thread_num = NumThreads / TileY * TileY;

  if (x < in_effective_thread_num) {
    // Read a tile from input using block.
    int x_i = x / TileY;
    int x_j = x % TileY;
    int input_ind = input_origin_block_flat_index + x_i * input_dims[2] + x_j;
    int input_inc = BlockReadRows * input_dims[2];

    if (full_tile) {
#pragma unroll
      for (int ind_i = x_i; ind_i < (TileX); ind_i += BlockReadRows) {
        tile_sm[ind_i][x_j] = input[input_ind];
        input_ind += input_inc;
      }
    } else {
      if (x_j < tile_width) {
#pragma unroll
        for (int ind_i = x_i; ind_i < (tile_height); ind_i += BlockReadRows) {
          tile_sm[ind_i][x_j] = input[input_ind];
          input_ind += input_inc;
        }
      }
    }
  }

  __syncthreads();

  // Store sm value back to out
  Index3 output_block_tile_index = {
      input_block_tile_index[0], input_block_tile_index[2],
      input_block_tile_index[1],
  };

  Index3 block_tile_index_in_output = {
      output_block_tile_index[0], output_block_tile_index[1] * TileY,
      output_block_tile_index[2] * TileX,
  };

  int output_origin_block_flat_index =
      FlatTensorIndex(block_tile_index_in_output, output_dims);

  constexpr int out_effective_thread_num = NumThreads / TileX * TileX;

  if (x < out_effective_thread_num) {
    int x_i = x / TileX;
    int x_j = x % TileX;
    int output_ind =
        output_origin_block_flat_index + x_i * output_dims[2] + x_j;
    int output_inc = BlockWriteRows * output_dims[2];

    if (full_tile) {
#pragma unroll
      for (int ind_i = x_i; ind_i < (TileY); ind_i += BlockWriteRows) {
        output[output_ind] = tile_sm[x_j][ind_i];
        output_ind += output_inc;
      }
    } else {
      if (x_j < tile_height) {
#pragma unroll
        for (int ind_i = x_i; ind_i < (tile_width); ind_i += BlockWriteRows) {
          output[output_ind] = tile_sm[x_j][ind_i];
          output_ind += output_inc;
        }
      }
    }
  }
}

// This function will find combination of long_side X short_side in backups
template <int TSIZE>
bool SelectProperTileSize(std::vector<std::pair<int, int>>* tiles) {
  PADDLE_ENFORCE_LE(
      TSIZE, 16,
      platform::errors::InvalidArgument(
          "The tile size should smaller than 16, but received is:%d.", TSIZE));

  PADDLE_ENFORCE_EQ(
      (TSIZE & (TSIZE - 1)), 0,
      platform::errors::InvalidArgument(
          "Data types should be powers of 2, but reived size is:%d.", TSIZE));

  const int kMaxLongSideLen = 1024;
  const int kMaxShortSideLen = 15;

  for (int long_side = 32; long_side <= kMaxLongSideLen; long_side *= 2) {
    for (int short_side = 2; short_side <= kMaxShortSideLen; short_side += 1) {
      if (CheckLongTileSize(long_side, short_side, TSIZE)) {
        tiles->push_back(std::make_pair(long_side, short_side));

        if (short_side == 2) return true;

        break;
      }
    }
  }
  return false;
}

// Use system built in type
template <int ByteSize>
struct SystemElemType;
template <>
struct SystemElemType<1> {
  using type = uint8_t;
};
template <>
struct SystemElemType<2> {
  using type = uint16_t;
};
template <>
struct SystemElemType<4> {
  using type = uint32_t;
};
template <>
struct SystemElemType<8> {
  using type = uint64_t;
};
template <>
struct SystemElemType<16> {
  using type = float4;
};

template <typename T, int tile_long, int tile_short>
void LaunchNarrowDims2TransposeKernel(const platform::CUDADeviceContext& d,
                                      int tile_size_i, int tile_size_j,
                                      int total_tiles_count, const T* input,
                                      const Dim3& input_dims, T* output) {
  constexpr int NumThreads = tile_long;
  if (tile_size_i <= tile_long && tile_size_j <= tile_short) {
    TilingSwapDim1And2<
        T, NumThreads, tile_long,
        tile_short><<<total_tiles_count, NumThreads, 0, d.stream()>>>(
        input, input_dims, output);
  } else {
    TilingSwapDim1And2<
        T, NumThreads, tile_short,
        tile_long><<<total_tiles_count, NumThreads, 0, d.stream()>>>(
        input, input_dims, output);
  }
}

template <typename T, int tile_long, int tile_short, typename dummy = void>
struct NarrowDims2TransposeDispatch {
  static void DoTranspose(const platform::CUDADeviceContext& d, int tile_size_i,
                          int tile_size_j, int total_tiles_count,
                          const T* input, const Dim3& input_dims, T* output) {
    PADDLE_ENFORCE_EQ(
        (tile_long & (tile_long - 1)), 0,
        platform::errors::InvalidArgument(
            "The length of the longer side of the tile should be power of 2."
            " But received value is:%d.",
            tile_long));

    bool request_satisfied = std::max(tile_size_i, tile_size_j) <= tile_long &&
                             std::min(tile_size_i, tile_size_j) <= tile_short;

    if (request_satisfied) {
      LaunchNarrowDims2TransposeKernel<T, tile_long, tile_short>(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
      return;
    }

    const bool long_side_request_not_satisfied =
        std::max(tile_size_i, tile_size_j) > tile_long;

    if (long_side_request_not_satisfied) {
      NarrowDims2TransposeDispatch<T, tile_long * 2, tile_short>::DoTranspose(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
    } else {
      NarrowDims2TransposeDispatch<T, tile_long, tile_short + 1>::DoTranspose(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
    }
  }
};

// If Not long tile size, goto this function when compile.
template <typename T, int tile_long, int tile_short>
struct NarrowDims2TransposeDispatch<
    T, tile_long, tile_short,
    typename std::enable_if<
        CheckNonLongTileSize(tile_long, tile_short, sizeof(T)), void>::type> {
  static void DoTranspose(const platform::CUDADeviceContext& d, int tile_size_i,
                          int tile_size_j, int total_tiles_count,
                          const T* input, const Dim3& input_dims, T* output) {
    PADDLE_ENFORCE_EQ(
        (tile_long & (tile_long - 1)), 0,
        platform::errors::InvalidArgument(
            "The length of the longer side of the tile should be power of 2."
            " But received value is:%d.",
            tile_long));

    bool request_satisfied = std::max(tile_size_i, tile_size_j) <= tile_long &&
                             std::min(tile_size_i, tile_size_j) <= tile_short;

    if (request_satisfied) {
      LaunchNarrowDims2TransposeKernel<T, tile_long, tile_short>(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
      return;
    }

    NarrowDims2TransposeDispatch<T, tile_long, tile_short + 1>::DoTranspose(
        d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
        output);
  }
};

// If long tile size, goto this function when compile.
template <typename T, int tile_long, int tile_short>
struct NarrowDims2TransposeDispatch<
    T, tile_long, tile_short,
    typename std::enable_if<CheckLongTileSize(tile_long, tile_short, sizeof(T)),
                            void>::type> {
  static void DoTranspose(const platform::CUDADeviceContext& d, int tile_size_i,
                          int tile_size_j, int total_tiles_count,
                          const T* input, const Dim3& input_dims, T* output) {
    PADDLE_ENFORCE_EQ(
        (tile_long & (tile_long - 1)), 0,
        platform::errors::InvalidArgument(
            "The length of the longer side of the tile should be power of 2,"
            " but received is:%d.",
            tile_long));

    LaunchNarrowDims2TransposeKernel<T, tile_long, tile_short>(
        d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
        output);
  }
};

template <typename T, bool conjugate = false>
void SwapDim1And2InNarrow(const platform::CUDADeviceContext& d, const T* input,
                          const Dim3& input_dims, T* output,
                          const int kMinTileSize) {
  // First get available tile sizes for the data type requested as backups
  std::vector<std::pair<int, int>> tile_sele;
  auto ret = SelectProperTileSize<sizeof(T)>(&tile_sele);
  PADDLE_ENFORCE_EQ(
      ret, true,
      platform::errors::InvalidArgument(
          "SelectProperTileSize should return true, but return value is:%d.",
          ret));

  int tile_long_edge = 0;
  int tile_short_edge = 0;
  float lowest_cost = std::numeric_limits<float>::max();
  int input_long_edge = std::max(input_dims[1], input_dims[2]);

  // Find the tile size that best suit in  inputs.
  for (auto tile_size_pair : tile_sele) {
    int proposed_tile_long_edge = tile_size_pair.first;
    // data may not aligned to tile, so some threads wasted, we need
    // to find least wasted threads, which means we need to find tile
    // can split input properly, in another words: num_wasted_threads=0.
    int num_wasted_threads = input_long_edge -
                             framework::CeilOrFloor<int, false>(
                                 input_long_edge, proposed_tile_long_edge) *
                                 proposed_tile_long_edge;

    int num_full_tiles = framework::CeilOrFloor<int, false>(
        input_long_edge, proposed_tile_long_edge);

    float cost = num_wasted_threads;

    if (cost <= lowest_cost) {
      tile_long_edge = proposed_tile_long_edge;
      tile_short_edge = tile_size_pair.second;
      lowest_cost = cost;
    }
    // break as we already find best tile size.
    if (cost == 0) break;
  }

  // The tile size we select should be match with input dim, long side to long
  // short side to short.
  // First set long side  as i if dim1 > Tile min size, then set dim2 as j.
  int select_tile_size_i =
      input_dims[1] >= kMinTileSize ? tile_long_edge : input_dims[1];
  int select_tile_size_j =
      input_dims[1] >= kMinTileSize ? input_dims[2] : tile_long_edge;

  // Check if i is long edge, if not set i as short.
  select_tile_size_i = select_tile_size_i == tile_long_edge
                           ? tile_long_edge
                           : std::min(select_tile_size_i, tile_short_edge);

  // Check if j is long edge, if not set j as short.
  select_tile_size_j = select_tile_size_j == tile_long_edge
                           ? tile_long_edge
                           : std::min(select_tile_size_j, tile_short_edge);

  // Here finally get proper long X short tile size.
  Dim3 input_dims_aligned = {
      input_dims[0],
      framework::CeilOrFloor<int, true>(input_dims[1], select_tile_size_i),
      framework::CeilOrFloor<int, true>(input_dims[2], select_tile_size_j),
  };

  int total_tiles_count =
      input_dims_aligned[0] * input_dims_aligned[1] * input_dims_aligned[2];

  // Suppose T can be replaced by system builtin types
  using ElemType = typename SystemElemType<sizeof(T)>::type;

  NarrowDims2TransposeDispatch<ElemType, 32, 2>::DoTranspose(
      d, select_tile_size_i, select_tile_size_j, total_tiles_count,
      reinterpret_cast<const ElemType*>(input), input_dims,
      reinterpret_cast<ElemType*>(output));
}

// This is for case that cannot do coalescing read and write.
// Or input is too small to split into tiles.
template <typename T, int pos0, int pos1, int pos2>
__global__ void TransposeSimpleKernel(int nthreads, const T* __restrict__ input,
                                      Dim3 input_dims, T* __restrict__ output) {
  Dim3 output_dims;
  output_dims[pos0] = input_dims[0];
  output_dims[pos1] = input_dims[1];
  output_dims[pos2] = input_dims[2];

  CUDA_1D_KERNEL_LOOP(output_index, nthreads) {
    Index3 output_tensor_index = ConvertTensorIndex(output_index, output_dims);

    Index3 input_tensor_index;
    input_tensor_index[0] = output_tensor_index[pos0];
    input_tensor_index[1] = output_tensor_index[pos1];
    input_tensor_index[2] = output_tensor_index[pos2];

    int input_index = FlatTensorIndex(input_tensor_index, input_dims);

    output[output_index] = input[input_index];
  }
}

// Here suppose convert all tensor to dim3, so just change dim1 and 2.
template <typename T>
void SendSwapDim1And2InTranspose(const platform::CUDADeviceContext& d,
                                 const T* input, const Dim3& input_dims,
                                 T* output) {
  // Suppose tile size > 16
  static const int kMinTileSize = 16;
  static const int kMinNarrowTileSize = 96;

  bool large_tile =
      input_dims[1] >= kMinTileSize && input_dims[2] >= kMinTileSize;
  bool narrow_tile = input_dims[1] >= kMinNarrowTileSize ||
                     input_dims[2] >= kMinNarrowTileSize;
  if (large_tile) {
    // If input is large square, such as 32X32, use SM to do copy.
    // suppose 32 X 32 gives best performance, and 8 warp in block.
    constexpr int kTileSize = 32;
    constexpr int kNumThreads = 256;

    Dim3 input_dims_aligned = {
        input_dims[0],
        framework::CeilOrFloor<int, true>(input_dims[1], kTileSize),
        framework::CeilOrFloor<int, true>(input_dims[2], kTileSize),
    };

    int total_tiles_count =
        input_dims_aligned[0] * input_dims_aligned[1] * input_dims_aligned[2];

    TilingSwapDim1And2<
        T, kNumThreads, kTileSize,
        kTileSize><<<total_tiles_count, kNumThreads, 0, d.stream()>>>(
        input, input_dims, output);

  } else if (narrow_tile) {
    // If input shape is like Rect, such as 2X100, use Narrow tile size.
    // It makes things complicated, because need to find a tile can coverr
    // input and also reach best coalescing.
    SwapDim1And2InNarrow<T>(d, input, input_dims, output, kMinTileSize);
  } else {
    // If input shape is small, such as 8X8, just do simple copy
    int total_elements = input_dims[0] * input_dims[1] * input_dims[2];
    auto config = GetGpuLaunchConfig1D(d, total_elements);
    TransposeSimpleKernel<T, 0, 2, 1><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, d.stream()>>>(
        total_elements, input, input_dims, output);
  }
}

template <typename T>
struct SwapDim1And2InTranspose {
  typedef platform::CUDADeviceContext Device;
  void operator()(const Device& d, const T* in,
                  const std::vector<int>& combined_dims, T* out) {
    Dim3 input_dims = {static_cast<int>(combined_dims[0]),
                       static_cast<int>(combined_dims[1]),
                       static_cast<int>(combined_dims[2])};
    SendSwapDim1And2InTranspose<T>(d, in, input_dims, out);
  }
};

template <typename T>
struct SwapDim0And2InTranspose {
  typedef platform::CUDADeviceContext Device;
  void operator()(const Device& d, const T* in,
                  const std::vector<int>& combined_dims, T* out) {
    Dim3 input_dims = {static_cast<int>(combined_dims[0]),
                       static_cast<int>(combined_dims[1]),
                       static_cast<int>(combined_dims[2])};

    size_t total_size = combined_dims[0] * combined_dims[1] * combined_dims[2];
    auto config = GetGpuLaunchConfig1D(d, total_size);

    TransposeSimpleKernel<T, 2, 1, 0><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, d.stream()>>>(
        total_size, in, input_dims, out);
  }
};

// This function is to combine dimension. fox example:
// (0, 1, 3, 2) --> (0, 2, 1)
inline void CombineTransposeDim3(const framework::DDim& shape,
                                 const std::vector<int>& perm,
                                 std::vector<int>* new_perm,
                                 framework::DDim* new_dims) {
  PADDLE_ENFORCE_EQ(shape.size(), perm.size(),
                    platform::errors::InvalidArgument(
                        " shape should have the save dim with perm, but"
                        " received shape size is:%d, perm size is:%d.",
                        shape.size(), perm.size()));

  std::vector<int> dim_vec;
  if (shape.size() == 1) {
    // If input dimension is already 1, no need to combine dim.
    new_perm->resize(1);
    (*new_perm)[0] = perm[0];
    dim_vec.push_back(shape[0]);
    *new_dims = framework::make_ddim(dim_vec);
    return;
  }
  std::vector<int> new_dim_pos(shape.size(), -1);
  std::vector<int> combined_dims(shape.size(), 0);
  int cur_head = perm[0];
  new_dim_pos[cur_head] = 0;
  combined_dims[0] = shape[cur_head];
  int dim_idx = 0;
  for (int perm_idx = 1; perm_idx < shape.size(); ++perm_idx) {
    // combine consecutive dimensions.
    if (cur_head + 1 == perm[perm_idx]) {
      cur_head = perm[perm_idx];
      combined_dims[dim_idx] *= shape[cur_head];
    } else {
      // Else start a new dimension.
      cur_head = perm[perm_idx];
      dim_idx++;
      new_dim_pos[cur_head] = dim_idx;
      combined_dims[dim_idx] = shape[cur_head];
    }
  }

  new_perm->resize(dim_idx + 1);

  dim_idx = 0;
  for (int i = 0; i < new_dim_pos.size(); ++i) {
    if (new_dim_pos[i] >= 0) {
      int new_perm_idx = new_dim_pos[i];
      (*new_perm)[dim_idx] = new_perm_idx;
      dim_vec.push_back(combined_dims[new_perm_idx]);
      dim_idx++;
    }
  }

  *new_dims = framework::make_ddim(dim_vec);
}

template <typename T>
struct TransposeSimple {
  static bool run(const platform::CUDADeviceContext& ctx, const Tensor& in,
                  const std::vector<int32_t> perm, Tensor* out) {
    // First reduce the dimensions of the input tensor if possible.
    std::vector<int> new_perm;
    framework::DDim new_dims;
    CombineTransposeDim3(in.dims(), perm, &new_perm, &new_dims);

    // Only use tile copy GPU kernel when dimension is 2 or 3.
    int dims = new_dims.size();
    std::vector<int> new_dim_vec = framework::vectorize<int>(new_dims);
    if (dims < 2 || dims > 3) return false;
    auto in_data = in.data<T>();
    auto out_data = out->data<T>();
    // In most cases, dim will not greater than 3 after combine.
    switch (dims) {
      case 2:
        if (new_perm[0] == 1 && new_perm[1] == 0) {
          // Add the first dimension size as 1.
          new_dim_vec.insert(new_dim_vec.begin(), 1);
          SwapDim1And2InTranspose<T>()(ctx, in_data, new_dim_vec, out_data);
          return true;
        }
        break;
      case 3:
        // In this case, suppose we can do coalescing read and write in tile.
        if (new_perm == std::vector<int>({0, 2, 1})) {
          SwapDim1And2InTranspose<T>()(ctx, in_data, new_dim_vec, out_data);
          return true;
        } else if (new_perm == std::vector<int>({2, 1, 0})) {
          // Maybe can optimize later, find a way to do coalescing memory copy.
          // But I think it depends on the data size. If span is not large,
          // maybe
          // can do coalescing.
          SwapDim0And2InTranspose<T>()(ctx, in_data, new_dim_vec, out_data);
          return true;
        } else {
          return false;
        }
        break;
      default:
        return false;
    }
    return false;
  }
};

template <typename DeviceContext, typename T>
class TransposeGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<framework::Tensor>("X");
    auto* out = context.Output<framework::Tensor>("Out");
    out->mutable_data<T>(context.GetPlace());
    if (out->numel() == 0) {
      return;
    }

    std::vector<int> axis = context.Attr<std::vector<int>>("axis");
    int ndims = axis.size();
    const auto& dev_ctx = context.template device_context<DeviceContext>();
    auto ret = TransposeSimple<T>::run(dev_ctx, *x, axis, out);
    if (!ret) {
      TransCompute<DeviceContext, T>(ndims, dev_ctx, *x, out, axis);
    }
  }
};
template <typename DeviceContext, typename T>
class TransposeGradGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* out_grad =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* x_grad =
        context.Output<framework::Tensor>(framework::GradVarName("X"));
    if (!x_grad) return;

    x_grad->mutable_data<T>(context.GetPlace());
    if (x_grad->numel() == 0) {
      return;
    }
    std::vector<int> axis = context.Attr<std::vector<int>>("axis");
    std::vector<int> reversed_axis(axis);

    for (size_t i = 0; i < axis.size(); i++) {
      reversed_axis[axis[i]] = i;
    }

    int ndims = axis.size();
    const auto& dev_ctx = context.template device_context<DeviceContext>();
    auto ret =
        TransposeSimple<T>::run(dev_ctx, *out_grad, reversed_axis, x_grad);
    if (!ret) {
      TransCompute<DeviceContext, T>(ndims, dev_ctx, *out_grad, x_grad,
                                     reversed_axis);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    transpose,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    transpose_grad,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext,
                                plat::float16>);

REGISTER_OP_CUDA_KERNEL(
    transpose2,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, int32_t>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    transpose2_grad,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, int32_t>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext,
                                plat::float16>);
