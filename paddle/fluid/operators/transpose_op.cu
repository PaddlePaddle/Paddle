#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <algorithm>
#include <limits>
#include <utility>

#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/gpu_launch_param_config.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

struct EqualTo {
  constexpr bool operator()(int a, int b) const { return a == b; }
};

struct GreaterThan {
  constexpr bool operator()(int a, int b) const { return a > b; }
};

// Value can be decided in compile time.
template <typename FUN, int INT_32 = 32>
constexpr bool CheckProperTileSize(int tile_long, int tile_short, int size_T,
                                   FUN op) {
  return (size_T == 16 && ((tile_long == INT_32 && op(tile_short, 4)) ||
                           (tile_long == 2 * INT_32 && op(tile_short, 4)) ||
                           (tile_long == 4 * INT_32 && op(tile_short, 4)) ||
                           (tile_long == 8 * INT_32 && op(tile_short, 2)))) ||
         (size_T == 8 && ((tile_long == INT_32 && op(tile_short, 15)) ||
                          (tile_long == 2 * INT_32 && op(tile_short, 15)) ||
                          (tile_long == 4 * INT_32 && op(tile_short, 8)) ||
                          (tile_long == 8 * INT_32 && op(tile_short, 4)) ||
                          (tile_long == 16 * INT_32 && op(tile_short, 2)))) ||
         ((size_T == 4 || size_T == 2 || size_T == 1) &&
          ((tile_long == INT_32 && op(tile_short, 15)) ||
           (tile_long == 2 * INT_32 && op(tile_short, 15)) ||
           (tile_long == 4 * INT_32 && op(tile_short, 8)) ||
           (tile_long == 8 * INT_32 && op(tile_short, 4)) ||
           (tile_long == 16 * INT_32 && op(tile_short, 2)) ||
           (tile_long == 16 * INT_32 && op(tile_short, 2))));
}

constexpr bool CheckLongTileSize(int tile_long, int tile_short, int size_T) {
  return CheckProperTileSize(tile_long, tile_short, size_T, EqualTo());
}

constexpr bool CheckOutsideTileSize(int tile_long, int tile_short, int size_T) {
  return CheckProperTileSize(tile_long, tile_short, size_T, GreaterThan());
}

constexpr bool CheckNonLongTileSize(int tile_long, int tile_short, int size_T) {
  return !CheckOutsideTileSize(tile_long, tile_short, size_T) &&
         (CheckOutsideTileSize(tile_long * 2, tile_short, size_T) ||
          CheckOutsideTileSize(tile_long, tile_short + 1, size_T)) &&
         !CheckLongTileSize(tile_long, tile_short, size_T);
}

template <typename T, T DefaultValue>
struct Array3 {
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE T& operator[](int index) {
    return data[index];
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE const T& operator[](int index) const {
    return data[index];
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Array3() {
    for (int i = 0; i < 3; i++) {
      data[i] = DefaultValue;
    }
  }
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Array3(T d0, T d1, T d2) {
    data[0] = d0;
    data[1] = d1;
    data[2] = d2;
  }
  EIGEN_STRONG_INLINE Array3(const std::array<T, 3>& array) {
    for (int i = 0; i < 3; i++) {
      data[i] = array[i];
    }
  }
  T data[3];
};

struct Dim3 : Array3<int, 1> {
  typedef Array3<int, 1> Base;
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Dim3() : Base() {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Dim3(int a0, int a1, int a2)
      : Base(a0, a1, a2) {}
  EIGEN_STRONG_INLINE Dim3(const std::array<int, 3>& array) : Base(array) {}
};

struct Index3 : Array3<int, 0> {
  typedef Array3<int, 0> Base;
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index3() : Base() {}
  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index3(int a0, int a1, int a2)
      : Base(a0, a1, a2) {}
};

// Flat index with real domension
EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE int FlatTensorIndex(const Index3& index,
                                                          const Dim3& dims) {
  int flat_index = index[0];
  for (int i = 1; i < 3; i++) {
    flat_index = flat_index * dims[i] + index[i];
  }
  return flat_index;
}

// Convert index to tensor index with dimension..
EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE Index3
ConvertTensorIndex(int index, const Dim3& dims) {
  Index3 tensor_index;
  for (int i = 2; i >= 0; i--) {
    int new_index = index / dims[i];
    tensor_index[i] = index - dims[i] * new_index;
    index = new_index;
  }
  return tensor_index;
}

// Use SM to do data transfer, load a tile into SM then store out.

template <typename T, int NumThreads, int TileX, int TileY>
__global__ void TilingSwapDim1And2(const T* __restrict__ input, Dim3 input_dims,
                                   T* __restrict__ output) {
  assert(blockDim.x == NumThreads);
  assert(blockDim.y == 1);
  assert(blockDim.z == 1);
  assert(gridDim.y == 1);
  assert(gridDim.z == 1);

  constexpr int BlockReadRows = NumThreads / TileY;
  constexpr int BlockWriteRows = NumThreads / TileX;

  // One extra line in the inner dimension to avoid share memory bank conflict.
  __shared__ __align__(
      alignof(T)) char share_mem_ptr[TileX * (TileY + 1) * sizeof(T)];
  typedef T(*ShareMemory)[TileY + 1];

  ShareMemory tile_sm = reinterpret_cast<ShareMemory>(share_mem_ptr);

  int x = threadIdx.x;

  Dim3 output_dims = {
      input_dims[0], input_dims[2], input_dims[1],
  };

  // Align dim to Tiles
  Dim3 tile_aligned_input_dim = {
      input_dims[0], (input_dims[1] + TileX - 1) / TileX,
      (input_dims[2] + TileY - 1) / TileY,
  };

  // Converts block idx to tile index, each block process a tile
  Index3 input_block_tile_index =
      ConvertTensorIndex(blockIdx.x, tile_aligned_input_dim);

  // Compute real index align to tile:0, 32, 64...
  Index3 block_tile_index_in_input = {
      input_block_tile_index[0], input_block_tile_index[1] * TileX,
      input_block_tile_index[2] * TileY,
  };

  // Compute block flat index against input dims.
  int input_origin_block_flat_index =
      FlatTensorIndex(block_tile_index_in_input, input_dims);

  bool full_tile = true;
  int tile_width = TileY;

  // Last row is not full.
  if (input_block_tile_index[2] == tile_aligned_input_dim[2] - 1) {
    tile_width = input_dims[2] - (tile_aligned_input_dim[2] - 1) * TileY;
    full_tile &= false;
  }

  int tile_height = TileX;

  if (input_block_tile_index[1] == tile_aligned_input_dim[1] - 1) {
    tile_height = input_dims[1] - (tile_aligned_input_dim[1] - 1) * TileX;
    full_tile &= false;
  }

  constexpr int in_effective_thread_num = NumThreads / TileY * TileY;

  if (x < in_effective_thread_num) {
    // Read a tile from input using block.
    int x_i = x / TileY;
    int x_j = x % TileY;
    int input_ind = input_origin_block_flat_index + x_i * input_dims[2] + x_j;
    int input_inc = BlockReadRows * input_dims[2];

    if (full_tile) {
#pragma unroll
      for (int ind_i = x_i; ind_i < (TileX); ind_i += BlockReadRows) {
        tile_sm[ind_i][x_j] = input[input_ind];
        input_ind += input_inc;
      }
    } else {
      if (x_j < tile_width) {
        for (int ind_i = x_i; ind_i < (tile_height); ind_i += BlockReadRows) {
          tile_sm[ind_i][x_j] = input[input_ind];
          input_ind += input_inc;
        }
      }
    }
  }

  __syncthreads();

  // Store sm value back to out
  Index3 output_block_tile_index = {
      input_block_tile_index[0], input_block_tile_index[2],
      input_block_tile_index[1],
  };

  Index3 block_tile_index_in_output = {
      output_block_tile_index[0], output_block_tile_index[1] * TileY,
      output_block_tile_index[2] * TileX,
  };

  int output_origin_block_flat_index =
      FlatTensorIndex(block_tile_index_in_output, output_dims);

  constexpr int out_effective_thread_num = NumThreads / TileX * TileX;

  if (x < out_effective_thread_num) {
    int x_i = x / TileX;
    int x_j = x % TileX;
    int output_ind =
        output_origin_block_flat_index + x_i * output_dims[2] + x_j;
    int output_inc = BlockWriteRows * output_dims[2];

    if (full_tile) {
#pragma unroll
      for (int ind_i = x_i; ind_i < (TileY); ind_i += BlockWriteRows) {
        output[output_ind] = tile_sm[x_j][ind_i];
        output_ind += output_inc;
      }
    } else {
      if (x_j < tile_height) {
        for (int ind_i = x_i; ind_i < (tile_width); ind_i += BlockWriteRows) {
          output[output_ind] = tile_sm[x_j][ind_i];
          output_ind += output_inc;
        }
      }
    }
  }
}

template <int TSIZE>
const std::vector<std::pair<int, int>>& SelectProperTileSize() {
  PADDLE_ENFORCE_LE(
      TSIZE, 16,
      "Currently, only data types of sizes 16 bytes or less are supported.");
  PADDLE_ENFORCE_EQ((TSIZE & (TSIZE - 1)), 0,
                    "Data types must have sizes that are powers of 2.");
  auto frontier = std::vector<std::pair<int, int>>();
  const int kMaxLongSideLen = 1024;
  const int kMaxShortSideLen = 15;

  for (int long_side = 32; long_side <= kMaxLongSideLen; long_side *= 2) {
    for (int short_side = 2; short_side <= kMaxShortSideLen; short_side += 1) {
      if (CheckLongTileSize(long_side, short_side, TSIZE)) {
        frontier.push_back(std::make_pair(long_side, short_side));

        if (short_side == 2) return frontier;

        break;
      }
    }
  }
  LOG(FATAL) << "The corresponding short side length of the largest long side "
                "length has to be 2.";
  return frontier;
}

// Get Transpose Type
template <int ElemBytes>
struct TransposeElemType;
template <>
struct TransposeElemType<1> {
  using type = uint8_t;
};
template <>
struct TransposeElemType<2> {
  using type = uint16_t;
};
template <>
struct TransposeElemType<4> {
  using type = uint32_t;
};
template <>
struct TransposeElemType<8> {
  using type = uint64_t;
};
template <>
struct TransposeElemType<16> {
  using type = float4;
};

template <typename IntegralType, bool ceil>
IntegralType CeilOrFloorOfRatio(IntegralType numerator,
                                IntegralType denominator) {
  PADDLE_ENFORCE_NE(0, denominator, "Division by zero is not supported.");

  const IntegralType rounded_toward_zero = numerator / denominator;
  const IntegralType intermediate_product = rounded_toward_zero * denominator;

  if (ceil) {
    const bool needs_adjustment =
        (rounded_toward_zero >= 0) &&
        ((denominator > 0 && numerator > intermediate_product) ||
         (denominator < 0 && numerator < intermediate_product));
    const IntegralType adjustment = static_cast<IntegralType>(needs_adjustment);
    const IntegralType ceil_of_ratio = rounded_toward_zero + adjustment;
    return ceil_of_ratio;
  } else {
    const bool needs_adjustment =
        (rounded_toward_zero <= 0) &&
        ((denominator > 0 && numerator < intermediate_product) ||
         (denominator < 0 && numerator > intermediate_product));
    const IntegralType adjustment = static_cast<IntegralType>(needs_adjustment);
    const IntegralType floor_of_ratio = rounded_toward_zero - adjustment;
    return floor_of_ratio;
  }
}

template <typename T, int tile_long, int tile_short>
void LaunchNarrowDims2TransposeKernel(const platform::CUDADeviceContext& d,
                                      int tile_size_i, int tile_size_j,
                                      int total_tiles_count, const T* input,
                                      const Dim3& input_dims, T* output) {
  constexpr int NumThreads = tile_long;
  if (tile_size_i <= tile_long && tile_size_j <= tile_short) {
    TilingSwapDim1And2<
        T, NumThreads, tile_long,
        tile_short><<<total_tiles_count, NumThreads, 0, d.stream()>>>(
        input, input_dims, output);
  } else {
    TilingSwapDim1And2<
        T, NumThreads, tile_short,
        tile_long><<<total_tiles_count, NumThreads, 0, d.stream()>>>(
        input, input_dims, output);
  }
}

template <typename T, int tile_long, int tile_short, typename dummy = void>
struct NarrowDims2TransposeDispatch {
  static void DoIt(const platform::CUDADeviceContext& d, int tile_size_i,
                   int tile_size_j, int total_tiles_count, const T* input,
                   const Dim3& input_dims, T* output) {
    PADDLE_ENFORCE_EQ(
        (tile_long & (tile_long - 1)), 0,
        "The length of the longer side of the tile is always a power of 2.");
    bool request_satisfied = std::max(tile_size_i, tile_size_j) <= tile_long &&
                             std::min(tile_size_i, tile_size_j) <= tile_short;

    if (request_satisfied) {
      LaunchNarrowDims2TransposeKernel<T, tile_long, tile_short>(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
      return;
    }

    const bool long_side_request_not_satisfied =
        std::max(tile_size_i, tile_size_j) > tile_long;

    if (long_side_request_not_satisfied) {
      NarrowDims2TransposeDispatch<T, tile_long * 2, tile_short>::DoIt(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
    } else {
      NarrowDims2TransposeDispatch<T, tile_long, tile_short + 1>::DoIt(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
    }
  }
};

template <typename T, int tile_long, int tile_short>
struct NarrowDims2TransposeDispatch<
    T, tile_long, tile_short,
    typename std::enable_if<
        CheckNonLongTileSize(tile_long, tile_short, sizeof(T)), void>::type> {
  static void DoIt(const platform::CUDADeviceContext& d, int tile_size_i,
                   int tile_size_j, int total_tiles_count, const T* input,
                   const Dim3& input_dims, T* output) {
    PADDLE_ENFORCE_EQ(
        (tile_long & (tile_long - 1)), 0,
        "The length of the longer side of the tile is always a power of 2.");
    bool request_satisfied = std::max(tile_size_i, tile_size_j) <= tile_long &&
                             std::min(tile_size_i, tile_size_j) <= tile_short;

    if (request_satisfied) {
      LaunchNarrowDims2TransposeKernel<T, tile_long, tile_short>(
          d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
          output);
      return;
    }

    NarrowDims2TransposeDispatch<T, tile_long, tile_short + 1>::DoIt(
        d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
        output);
  }
};

template <typename T, int tile_long, int tile_short>
struct NarrowDims2TransposeDispatch<
    T, tile_long, tile_short,
    typename std::enable_if<CheckLongTileSize(tile_long, tile_short, sizeof(T)),
                            void>::type> {
  static void DoIt(const platform::CUDADeviceContext& d, int tile_size_i,
                   int tile_size_j, int total_tiles_count, const T* input,
                   const Dim3& input_dims, T* output) {
    PADDLE_ENFORCE_EQ(
        (tile_long & (tile_long - 1)), 0,
        "The length of the longer side of the tile is always a power of 2.");

    LaunchNarrowDims2TransposeKernel<T, tile_long, tile_short>(
        d, tile_size_i, tile_size_j, total_tiles_count, input, input_dims,
        output);
  }
};

template <typename T, bool conjugate = false>
void SwapDim1And2InNarrow(const platform::CUDADeviceContext& d, const T* input,
                          const Dim3& input_dims, T* output,
                          const int kMinTileSize) {
  // Get available tile sizes here for the data type requested:
  const auto& tile_spec = SelectProperTileSize<sizeof(T)>();

  int tile_long_edge = 0;
  int tile_short_edge = 0;
  float lowest_cost = std::numeric_limits<float>::max();
  int input_long_edge = std::max(input_dims[1], input_dims[2]);

  for (auto tile_size_pair : tile_spec) {
    int proposed_tile_long_edge = tile_size_pair.first;
    // data may not aligned to tile, so some threads wasted
    int num_wasted_threads = input_long_edge -
                             CeilOrFloorOfRatio<int, false>(
                                 input_long_edge, proposed_tile_long_edge) *
                                 proposed_tile_long_edge;

    int num_full_tiles = CeilOrFloorOfRatio<int, false>(
        input_long_edge, proposed_tile_long_edge);

    float cost = 0;

    if (num_full_tiles <= 1) cost = num_wasted_threads;

    // Find least weasted threads.
    if (cost <= lowest_cost) {
      tile_long_edge = proposed_tile_long_edge;
      tile_short_edge = tile_size_pair.second;
      lowest_cost = cost;
    }
  }

  // The tile size we select should be match with input dim, long side to long
  // short side to short.
  // First set long side  as i if dim1 > Tile min size, then set dim2 as j.
  int select_tile_size_i =
      input_dims[1] >= kMinTileSize ? tile_long_edge : input_dims[1];
  int select_tile_size_j =
      input_dims[1] >= kMinTileSize ? input_dims[2] : tile_long_edge;

  // Check if i is long edge, if not set i as short.
  select_tile_size_i = select_tile_size_i == tile_long_edge
                           ? tile_long_edge
                           : std::min(select_tile_size_i, tile_short_edge);

  // Check if j is long edge, if not set j as short.
  select_tile_size_j = select_tile_size_j == tile_long_edge
                           ? tile_long_edge
                           : std::min(select_tile_size_j, tile_short_edge);

  Dim3 input_dims_aligned = {
      input_dims[0],
      CeilOrFloorOfRatio<int, true>(input_dims[1], select_tile_size_i),
      CeilOrFloorOfRatio<int, true>(input_dims[2], select_tile_size_j),
  };

  int total_tiles_count =
      input_dims_aligned[0] * input_dims_aligned[1] * input_dims_aligned[2];

  using ElemType = typename TransposeElemType<sizeof(T)>::type;
  static_assert(alignof(T) >= alignof(ElemType), "Unexpected data alignment.");
  NarrowDims2TransposeDispatch<ElemType, 32, 2>::DoIt(
      d, select_tile_size_i, select_tile_size_j, total_tiles_count,
      reinterpret_cast<const ElemType*>(input), input_dims,
      reinterpret_cast<ElemType*>(output));
}

template <typename T, int pos0, int pos1, int pos2>
__global__ void TransposeSimpleKernel(int nthreads, const T* __restrict__ input,
                                      Dim3 input_dims, T* __restrict__ output) {
  Dim3 output_dims;
  output_dims[pos0] = input_dims[0];
  output_dims[pos1] = input_dims[1];
  output_dims[pos2] = input_dims[2];

  CUDA_1D_KERNEL_LOOP(output_index, nthreads) {
    Index3 output_tensor_index = ConvertTensorIndex(output_index, output_dims);

    Index3 input_tensor_index;
    input_tensor_index[0] = output_tensor_index[pos0];
    input_tensor_index[1] = output_tensor_index[pos1];
    input_tensor_index[2] = output_tensor_index[pos2];

    int input_index = FlatTensorIndex(input_tensor_index, input_dims);

    output[output_index] = input[input_index];
  }
}

// Here suppose convert all tensor to dim3, so just change dim1 and 2.
template <typename T>
void RunSwapDim1And2InTranspose(const platform::CUDADeviceContext& d,
                                const T* input, const Dim3 input_dims,
                                T* output) {
  // Suppose tile size > 16
  static const int kMinTileSize = 16;
  static const int kMinNarrowTileSize = 96;

  bool large_tile =
      input_dims[1] >= kMinTileSize && input_dims[2] >= kMinTileSize;
  bool narrow_tile = input_dims[1] >= kMinNarrowTileSize ||
                     input_dims[2] >= kMinNarrowTileSize;
  if (large_tile) {
    // suppose 32 X 32 gives best performance, and 8 warp in block.
    constexpr int kTileSize = 32;
    constexpr int kNumThreads = 256;

    Dim3 input_dims_aligned = {
        input_dims[0], CeilOrFloorOfRatio<int, true>(input_dims[1], kTileSize),
        CeilOrFloorOfRatio<int, true>(input_dims[2], kTileSize),
    };

    int total_tiles_count =
        input_dims_aligned[0] * input_dims_aligned[1] * input_dims_aligned[2];

    TilingSwapDim1And2<
        T, kNumThreads, kTileSize,
        kTileSize><<<total_tiles_count, kNumThreads, 0, d.stream()>>>(
        input, input_dims, output);

  } else if (narrow_tile) {
    SwapDim1And2InNarrow<T>(d, input, input_dims, output, kMinTileSize);
  } else {
    int total_elements = input_dims[0] * input_dims[1] * input_dims[2];
    auto config = GetGpuLaunchConfig1D(d, total_elements);
    TransposeSimpleKernel<T, 0, 2, 1><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, d.stream()>>>(
        total_elements, input, input_dims, output);
  }
}

template <typename T>
struct SwapDim1And2InTranspose {
  typedef platform::CUDADeviceContext Device;
  void operator()(const Device& d, const T* in,
                  const std::vector<int>& combined_dims, T* out) {
    Dim3 input_dims = {static_cast<int>(combined_dims[0]),
                       static_cast<int>(combined_dims[1]),
                       static_cast<int>(combined_dims[2])};
    RunSwapDim1And2InTranspose<T>(d, in, input_dims, out);
  }
};

template <typename T>
struct SwapDim0And2InTranspose {
  typedef platform::CUDADeviceContext Device;
  void operator()(const Device& d, const T* in,
                  const std::vector<int>& combined_dims, T* out) {
    Dim3 input_dims = {static_cast<int>(combined_dims[0]),
                       static_cast<int>(combined_dims[1]),
                       static_cast<int>(combined_dims[2])};

    size_t total_size = combined_dims[0] * combined_dims[1] * combined_dims[2];
    auto config = GetGpuLaunchConfig1D(d, total_size);

    TransposeSimpleKernel<T, 2, 1, 0><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, d.stream()>>>(
        total_size, in, input_dims, out);
  }
};

inline void CombineTransposeDim3(const framework::DDim& shape,
                                 const std::vector<int>& perm,
                                 std::vector<int>* new_perm,
                                 framework::DDim* new_dims) {
  PADDLE_ENFORCE_EQ(shape.size(), perm.size(),
                    " shape should have the save dim with");
  std::vector<int> dim_vec;
  if (shape.size() == 1) {
    // If input dimension is already 1, n<<dim_idx;o need to reduce dimension.
    new_perm->resize(1);
    (*new_perm)[0] = perm[0];
    dim_vec.push_back(shape[0]);
    *new_dims = framework::make_ddim(dim_vec);
    return;
  }
  std::vector<int> new_dim_pos(shape.size(), -1);
  std::vector<int> combined_dims(shape.size(), 0);
  int cur_head = perm[0];
  new_dim_pos[cur_head] = 0;
  combined_dims[0] = shape[cur_head];
  int dim_idx = 0;
  for (int perm_idx = 1; perm_idx < shape.size(); ++perm_idx) {
    // combine consecutive dimensions.
    if (cur_head + 1 == perm[perm_idx]) {
      cur_head = perm[perm_idx];
      combined_dims[dim_idx] *= shape[cur_head];
    } else {
      // Else start a new dimension.
      cur_head = perm[perm_idx];
      dim_idx++;
      new_dim_pos[cur_head] = dim_idx;
      combined_dims[dim_idx] = shape[cur_head];
    }
  }

  new_perm->resize(dim_idx + 1);

  dim_idx = 0;
  for (int i = 0; i < new_dim_pos.size(); ++i) {
    if (new_dim_pos[i] >= 0) {
      int new_perm_idx = new_dim_pos[i];
      (*new_perm)[dim_idx] = new_perm_idx;
      dim_vec.push_back(combined_dims[new_perm_idx]);
      dim_idx++;
    }
  }

  *new_dims = framework::make_ddim(dim_vec);
}

template <typename T>
struct TransposeSimple {
  static bool run(const platform::CUDADeviceContext& ctx, const Tensor& in,
                  const std::vector<int32_t> perm, Tensor* out) {
    // First try to reduce the dimensions of the input tensor.
    std::vector<int> new_perm;
    framework::DDim new_dims;
    CombineTransposeDim3(in.dims(), perm, &new_perm, &new_dims);

    // Only use special GPU kernel when dimension is 2 or 3.
    int dims = new_dims.size();
    std::vector<int> new_dim_vec = framework::vectorize<int>(new_dims);
    if (dims < 2 || dims > 3) return false;
    auto in_data = in.data<T>();
    auto out_data = out->data<T>();

    switch (dims) {
      case 2:
        if (new_perm[0] == 1 && new_perm[1] == 0) {
          // Add the first dimension size as 1.
          new_dim_vec.insert(new_dim_vec.begin(), 1);
          SwapDim1And2InTranspose<T>()(ctx, in_data, new_dim_vec, out_data);
          return true;
        }
        break;
      case 3:
        if (new_perm == std::vector<int>({0, 2, 1})) {
          SwapDim1And2InTranspose<T>()(ctx, in_data, new_dim_vec, out_data);
          return true;
        } else if (new_perm == std::vector<int>({2, 1, 0})) {
          SwapDim0And2InTranspose<T>()(ctx, in_data, new_dim_vec, out_data);
          return true;
        } else {
          return false;
        }
        break;
      default:
        return false;
    }
    return false;
  }
};

template <typename DeviceContext, typename T>
class TransposeGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<framework::Tensor>("X");
    auto* out = context.Output<framework::Tensor>("Out");
    out->mutable_data<T>(context.GetPlace());

    std::vector<int> axis = context.Attr<std::vector<int>>("axis");
    int ndims = axis.size();
    const auto& dev_ctx = context.template device_context<DeviceContext>();
    auto ret = TransposeSimple<T>::run(dev_ctx, *x, axis, out);
    if (!ret) {
      TransCompute<DeviceContext, T>(ndims, dev_ctx, *x, out, axis);
    }
  }
};
template <typename DeviceContext, typename T>
class TransposeGradGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* out_grad =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* x_grad =
        context.Output<framework::Tensor>(framework::GradVarName("X"));
    if (!x_grad) return;

    x_grad->mutable_data<T>(context.GetPlace());
    std::vector<int> axis = context.Attr<std::vector<int>>("axis");
    std::vector<int> reversed_axis(axis);

    for (size_t i = 0; i < axis.size(); i++) {
      reversed_axis[axis[i]] = i;
    }

    int ndims = axis.size();
    const auto& dev_ctx = context.template device_context<DeviceContext>();
    auto ret =
        TransposeSimple<T>::run(dev_ctx, *out_grad, reversed_axis, x_grad);
    if (!ret) {
      TransCompute<DeviceContext, T>(ndims, dev_ctx, *out_grad, x_grad,
                                     reversed_axis);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    transpose,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    transpose_grad,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext,
                                plat::float16>);

REGISTER_OP_CUDA_KERNEL(
    transpose2,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, int32_t>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGPUKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    transpose2_grad,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, int32_t>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext, double>,
    ops::TransposeGradGPUKernel<paddle::platform::CUDADeviceContext,
                                plat::float16>);
