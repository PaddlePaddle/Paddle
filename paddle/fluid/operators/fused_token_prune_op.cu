#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <limits>

#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/phi/backends/gpu/gpu_launch_config.h"

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/fused_token_prune_op.cu.h"

namespace paddle {
namespace operators {

using framework::Tensor;

template <typename T>
struct AttnMaskFunctor {
  inline HOSTDEVICE T operator()(const T a, const T b) const {
    return b >= 0 ? a : 0;
  }
};

__global__ void FillIndex(int64_t* indices, int num_raws, int num_cols) {
  int num_threads = num_raws * num_cols;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < num_threads; tid += stride) {
    int col = tid % num_cols;
    indices[tid] = (int64_t)col;
  }
}

template <typename T>
__global__ void TakeAlongAxis(const T* src,
                              T* dst,
                              int64_t* indices,
                              int num_raws,
                              int src_num_cols,
                              int dst_num_cols,
                              int num_elements) {
  int num_threads = num_raws * dst_num_cols;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < num_threads; tid += stride) {
    int raw = tid / dst_num_cols;
    int col = tid % dst_num_cols;
    for (int i = 0; i < num_elements; ++i) {
      dst[tid * num_elements + i] =
          *(src + (raw * src_num_cols + indices[tid]) * num_elements + i);
    }
  }
}

template <typename T>
__global__ void MaximumFirst(T* mat, int num_raws, int num_cols, T max_value) {
  int num_threads = num_raws;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < num_threads; tid += stride) {
    mat[tid * num_cols] = max_value;
  }
}

template <typename T>
class FusedTokenPruneOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto& dev_ctx = context.cuda_device_context();
    // Inouts
    const Tensor* attn = context.Input<Tensor>("Attn");
    const Tensor* x = context.Input<Tensor>("X");
    const Tensor* mask = context.Input<Tensor>("Mask");
    const Tensor* new_mask = context.Input<Tensor>("NewMask");

    // Input dims
    auto attn_dims = attn->dims();
    auto x_dims = x->dims();
    auto new_mask_dims = new_mask->dims();

    auto bsz = attn_dims[0];
    auto num_heads = attn_dims[1];
    auto max_seq_len = attn_dims[2];
    auto c = x_dims[2];
    int slimmed_x_len = new_mask_dims[2];

    // Attrs
    const bool keep_first_token = context.Attr<bool>("keep_first_token");
    const bool keep_order = context.Attr<bool>("keep_order");

    // Outputs
    Tensor* out_slimmed_x = context.Output<Tensor>("SlimmedX");
    Tensor* slimmed_indices = context.Output<Tensor>("CLSInds");
    auto* out_slimmed_x_data =
        out_slimmed_x->mutable_data<T>(context.GetPlace());
    auto* slimmed_indices_data =
        slimmed_indices->mutable_data<int64_t>(context.GetPlace());

    // Intermediate variable
    Tensor attn_tmp;
    auto* attn_tmp_data =
        attn_tmp.mutable_data<T>(attn_dims, context.GetPlace());
    Tensor attn_accu;
    auto* attn_accu_data =
        attn_accu.mutable_data<T>({bsz, max_seq_len}, context.GetPlace());
    Tensor attn_accu_indices;
    auto* attn_accu_indices_data = attn_accu_indices.mutable_data<int64_t>(
        {bsz, max_seq_len}, context.GetPlace());
    Tensor sort_attn_accu;
    auto* sort_attn_accu_data =
        sort_attn_accu.mutable_data<T>({bsz, max_seq_len}, context.GetPlace());
    Tensor sort_attn_accu_indices;
    auto* sort_attn_accu_indices_data =
        sort_attn_accu_indices.mutable_data<int64_t>({bsz, max_seq_len},
                                                     context.GetPlace());
    Tensor temp_storage;

    // 1. Filter attn by mask
    std::vector<const Tensor*> ins;
    std::vector<Tensor*> outs;
    ins.emplace_back(attn);
    ins.emplace_back(mask);
    outs.emplace_back(&attn_tmp);
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        dev_ctx, ins, &outs, -1, AttnMaskFunctor<T>());

    // 2. Reduce sum
    const std::vector<int64_t> reduce_dims{1, 2};
    phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(dev_ctx,
                                                          attn_tmp,
                                                          false,
                                                          reduce_dims,
                                                          false,
                                                          attn_accu.dtype(),
                                                          &attn_accu);
    // 3. Prepare token indices
    phi::backends::gpu::GpuLaunchConfig config =
        phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, bsz * max_seq_len);
    FillIndex<<<config.block_per_grid,
                config.thread_per_block,
                0,
                dev_ctx.stream()>>>(attn_accu_indices_data, bsz, max_seq_len);

    // 4. Sort token indices by attn
    if (keep_first_token) {
      T max = std::numeric_limits<T>::max();
      config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, bsz);
      MaximumFirst<T>
          <<<config.block_per_grid,
             config.thread_per_block,
             0,
             dev_ctx.stream()>>>(attn_accu_data, bsz, max_seq_len, max);
    }
    size_t temp_storage_bytes = -1;
    int num_items = bsz * max_seq_len;
    int num_segments = bsz;

    hipcub::CountingInputIterator<int64_t> counting_iter(0);
    hipcub::TransformInputIterator<int64_t,
                                SegmentOffsetIter,
                                hipcub::CountingInputIterator<int64_t>>
        segment_offsets_t(counting_iter, SegmentOffsetIter(max_seq_len));
    // Determine temporary device storage requirements
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
            nullptr,
            temp_storage_bytes,
            attn_accu_data,
            sort_attn_accu_data,
            attn_accu_indices_data,
            sort_attn_accu_indices_data,
            num_items,
            num_segments,
            segment_offsets_t,
            segment_offsets_t + 1,
            0,
            sizeof(T) * 8,
            dev_ctx.stream()));
    // Allocate temporary storage
    int64_t temp_size = temp_storage_bytes;
    auto* temp_storage_data =
        temp_storage.mutable_data<uint8_t>({temp_size}, context.GetPlace());
    // Run sorting operation
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
            temp_storage_data,
            temp_storage_bytes,
            attn_accu_data,
            sort_attn_accu_data,
            attn_accu_indices_data,
            sort_attn_accu_indices_data,
            num_items,
            num_segments,
            segment_offsets_t,
            segment_offsets_t + 1,
            0,
            sizeof(T) * 8,
            dev_ctx.stream()));
    // 5. Slice
    auto slimmed_indices_tmp =
        phi::funcs::Slice<int64_t>(dev_ctx,
                                   sort_attn_accu_indices,
                                   {1} /*axes*/,
                                   {0} /*starts*/,
                                   {slimmed_x_len} /*ends*/);
    if (keep_order) {
      // 6. reorder
      num_items = bsz * slimmed_x_len;
      temp_storage_bytes = -1;
      hipcub::TransformInputIterator<int64_t,
                                  SegmentOffsetIter,
                                  hipcub::CountingInputIterator<int64_t>>
          segment_offsets_t2(counting_iter, SegmentOffsetIter(slimmed_x_len));
      PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
          nullptr,
          temp_storage_bytes,
          static_cast<int64_t*>(slimmed_indices_tmp.data()),
          static_cast<int64_t*>(slimmed_indices->data()),
          num_items,
          num_segments,
          segment_offsets_t2,
          segment_offsets_t2 + 1,
          0,
          sizeof(int64_t) * 8,
          dev_ctx.stream()));
      temp_size = temp_storage_bytes;
      temp_storage.Resize({temp_size});
      temp_storage_data =
          temp_storage.mutable_data<uint8_t>(context.GetPlace());
      PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
          temp_storage_data,
          temp_storage_bytes,
          static_cast<int64_t*>(slimmed_indices_tmp.data()),
          static_cast<int64_t*>(slimmed_indices->data()),
          num_items,
          num_segments,
          segment_offsets_t2,
          segment_offsets_t2 + 1,
          0,
          sizeof(int64_t) * 8,
          dev_ctx.stream()));
    } else {
      framework::TensorCopy(
          slimmed_indices_tmp, context.GetPlace(), slimmed_indices);
    }
    // 7. Get slimmed X by indices
    config =
        phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, bsz * slimmed_x_len);
    TakeAlongAxis<T><<<config.block_per_grid,
                       config.thread_per_block,
                       0,
                       dev_ctx.stream()>>>(x->data<T>(),
                                           out_slimmed_x_data,
                                           slimmed_indices->data<int64_t>(),
                                           bsz,
                                           max_seq_len,
                                           slimmed_x_len,
                                           c);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(fused_token_prune,
                        ops::FusedTokenPruneOpCUDAKernel<float>,
                        ops::FusedTokenPruneOpCUDAKernel<double>);
