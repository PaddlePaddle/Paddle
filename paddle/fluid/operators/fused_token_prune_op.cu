#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/sort.h>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"

#include "paddle/fluid/operators/detection/nms_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/phi/kernels/funcs/slice.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

namespace paddle {
namespace operators {

using framework::Tensor;

template <typename T>
struct AttnMaskFunctor {
  inline HOSTDEVICE T operator()(const T a, const T b) const {
    return b >= 0 ? a : 0;
  }
};

static __global__ void FillIndex(int* indices, int num_rows, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_rows * num_cols) return;

  int col = tid % num_cols;
  int row = tid / num_cols;

  // for (int j = row; j < num_rows; j += num_cols) {
  //     for (int i = col; i < num_cols; i += num_rows) {
  //     indices[j * num_cols + i] = i;
  //     }
  // }
  indices[tid] = col;
}

template <typename T>
static __global__ void SlicedArgsort(T* data, int* indices, int num_rows,
                                     int num_cols) {
  auto raw = blockIdx.x * blockDim.x + threadIdx.x;
  if (raw >= num_rows) return;
  thrust::sort_by_key(thrust::seq, data + raw * num_cols + 1,
                      data + (raw + 1) * num_cols, indices + raw * num_cols + 1,
                      thrust::greater<T>());
}

template <typename T>
static __global__ void TakeAlongAxis(const T* src, T* dst, int* indices,
                                     int num_rows, int src_num_cols,
                                     int dst_num_cols, int num_elements) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_rows * dst_num_cols) return;

  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  for (int i = 0; i < num_elements; ++i) {
    dst[tid * num_elements + i] =
        *(src + (raw * src_num_cols + indices[tid]) * num_elements + i);
  }
}

template <typename T>
class FusedTokenPruneOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    const Tensor* attn = context.Input<Tensor>("Attn");
    const Tensor* x = context.Input<Tensor>("X");
    const Tensor* mask = context.Input<Tensor>("Mask");
    Tensor* out_slimmed_x = context.Output<Tensor>("SlimmedX");
    auto factor = context.template Attr<float>("factor");
    auto* out_slimmed_x_data =
        out_slimmed_x->mutable_data<T>(context.GetPlace());

    Tensor attn_tmp;
    auto attn_dims = attn->dims();
    attn_tmp.Resize(attn_dims);
    auto* attn_tmp_data = attn_tmp.mutable_data<T>(context.GetPlace());

    std::vector<const Tensor*> ins;
    std::vector<Tensor*> outs;
    ins.emplace_back(attn);
    ins.emplace_back(mask);
    outs.emplace_back(&attn_tmp);
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        context.cuda_device_context(), ins, &outs, -1, AttnMaskFunctor<T>());

    // VLOG(4) << "attn after mask = " << attn_tmp;
    Tensor attn_by;
    attn_by.Resize({attn_dims[0], attn_dims[3]});
    auto* attn_by_data = attn_by.mutable_data<T>(context.GetPlace());
    const std::vector<int64_t> reduce_dims{1, 2};
    phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
        context.cuda_device_context(), attn_tmp, false, reduce_dims, false,
        attn_by.dtype(), &attn_by);

    // VLOG(4) << "attn after sum reduce = " << attn_by;
    // Tensor sliced_attn_by = phi::Slice<T, platform::CUDADeviceContext>(
    //     context.cuda_device_context(),
    //     attn_by, {1}, {1}, {attn_dims[3]});

    // Tensor *sorted_sliced_attn_by, *sorted_sliced_indices;
    // phi::ArgsortKernel<T, platform::CUDADeviceContext>(
    //     context.cuda_device_context(),
    //     sliced_attn_by, -1, true, sorted_sliced_attn_by,
    //     sorted_sliced_indices);

    Tensor attn_by_indices;
    attn_by_indices.Resize(attn_by.dims());
    auto* attn_by_indices_data =
        attn_by_indices.mutable_data<int>(context.GetPlace());

    auto ComputeBlockSize = [](int col) {
      if (col > 512)
        return 1024;
      else if (col > 256 && col <= 512)
        return 512;
      else if (col > 128 && col <= 256)
        return 256;
      else if (col > 64 && col <= 128)
        return 128;
      else
        return 64;
    };

    int grid_size = attn_dims[0], block_size = ComputeBlockSize(attn_dims[3]);
    FillIndex<<<grid_size, block_size, 0,
                context.cuda_device_context().stream()>>>(
        attn_by_indices_data, attn_dims[0], attn_dims[3]);

    // VLOG(4) << "before argsort attn indices = " << attn_by_indices;
    SlicedArgsort<
        T><<<grid_size, 1, 0, context.cuda_device_context().stream()>>>(
        attn_by_data, attn_by_indices_data, attn_dims[0], attn_dims[3]);
    // VLOG(4) << "after argsort attn indices = " << attn_by_indices;

    int slimmed_x_len = attn_dims[3] * factor;
    Tensor slimmed_indices =
        phi::funcs::Slice<int>(context.cuda_device_context(), attn_by_indices,
                               {1}, {0}, {slimmed_x_len});
    // VLOG(4) << "after slice attn indices = " << slimmed_indices;

    auto x_dims = x->dims();
    block_size = ComputeBlockSize(slimmed_x_len);
    TakeAlongAxis<T><<<grid_size, block_size, 0,
                       context.cuda_device_context().stream()>>>(
        x->data<T>(), out_slimmed_x_data, slimmed_indices.data<int>(),
        attn_dims[0], attn_dims[3], slimmed_x_len, x_dims[2]);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(fused_token_prune,
                        ops::FusedTokenPruneOpCUDAKernel<float>,
                        ops::FusedTokenPruneOpCUDAKernel<double>);
