#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/argsort_op.h"
#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"

#ifdef __HIPCC__
namespace rocprim {
namespace detail {
template <>
struct radix_key_codec_base<paddle::platform::float16>
    : radix_key_codec_integral<paddle::platform::float16, uint16_t> {};
}  // namespace detail
}  // namespace rocprim
#else
// set cub base traits in order to handle float16
namespace cub {
template <>
struct NumericTraits<paddle::platform::float16>
    : BaseTraits<FLOATING_POINT, true, false, uint16_t,
                 paddle::platform::float16> {};
}  // namespace cub
#endif

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

// Iter for move to next row
struct SegmentOffsetIter {
  EIGEN_DEVICE_FUNC
  explicit SegmentOffsetIter(int num_cols) : num_cols_(num_cols) {}

  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE int operator()(int idx) const {
    return idx * num_cols_;
  }

  int num_cols_;
};

template <typename T>
static __global__ void FillIndex(T* indices, T num_rows, T num_cols) {
  int col_id = threadIdx.x;
  int row_id = blockIdx.x;

  for (T j = row_id; j < num_rows; j += gridDim.x) {
    for (T i = col_id; i < num_cols; i += blockDim.x) {
      indices[j * num_cols + i] = i;
    }
  }
}

template <typename T, typename IndType>
static __global__ void FillFlattenGrad(const T* dO, const IndType* indices,
                                       int64_t size, T* dX) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < size; i += stride) {
    dX[indices[i]] = dO[i];
  }
}

template <typename T, typename IndType>
static __global__ void FillGrad(const T* dO, const IndType* indices, T* dX,
                                IndType num_rows, IndType num_cols) {
  int col_id = threadIdx.x;
  int row_id = blockIdx.x;

  for (IndType j = row_id; j < num_rows; j += gridDim.x) {
    for (IndType i = col_id; i < num_cols; i += blockDim.x) {
      dX[j * num_cols + indices[j * num_cols + i]] = dO[j * num_cols + i];
    }
  }
}

// Sort by flag descending, True: descending. False: Ascending.
// Default is false.
template <typename T, typename IndType>
void ArgFullSort(const platform::CUDADeviceContext& ctx, const Tensor* input,
                 Tensor* output, Tensor* indices, const IndType num_rows,
                 const IndType num_cols, const bool descending) {
  auto cu_stream = ctx.stream();

  Tensor input_indices;

  const std::vector<IndType> dims = {num_rows, num_cols};
  auto dim = framework::make_ddim(dims);
  input_indices.Resize(dim);
  input_indices.mutable_data<IndType>(ctx.GetPlace());

  size_t temp_storage_bytes = -1;

  auto ComputeBlockSize = [](IndType col) {
    if (col > 512)
      return 1024;
    else if (col > 256 && col <= 512)
      return 512;
    else if (col > 128 && col <= 256)
      return 256;
    else if (col > 64 && col <= 128)
      return 128;
    else
      return 64;
  };

  int block_size = ComputeBlockSize(num_cols);

  int maxGridDimX = ctx.GetCUDAMaxGridDimSize().x;
  // actually, int num_rows < max_grid_size
  int grid_size = num_rows < maxGridDimX ? num_rows : maxGridDimX;
  // Init a index array
  FillIndex<<<grid_size, block_size, 0, cu_stream>>>(
      input_indices.data<IndType>(), num_rows, num_cols);

  T* sorted_out_ptr;
  IndType* sorted_indices_ptr;

  const T* inp = input->data<T>();
  T* out = output->mutable_data<T>(ctx.GetPlace());
  IndType* ind = indices->mutable_data<IndType>(ctx.GetPlace());

  sorted_out_ptr = out;
  sorted_indices_ptr = ind;

  // create iter for counting input
  hipcub::CountingInputIterator<IndType> counting_iter(0);
  // segment_offset is used for move to next row
  hipcub::TransformInputIterator<IndType, SegmentOffsetIter,
                              hipcub::CountingInputIterator<IndType>>
      segment_offsets_t(counting_iter, SegmentOffsetIter(num_cols));

  gpuError_t err;
  if (descending) {
    err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        nullptr, temp_storage_bytes, inp, sorted_out_ptr,
        input_indices.data<IndType>(), sorted_indices_ptr, num_cols * num_rows,
        num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
        cu_stream);
  } else {
    err = hipcub::DeviceSegmentedRadixSort::SortPairs(
        nullptr, temp_storage_bytes, inp, sorted_out_ptr,
        input_indices.data<IndType>(), sorted_indices_ptr, num_cols * num_rows,
        num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
        cu_stream);
  }
  PADDLE_ENFORCE_CUDA_SUCCESS(err);

  Tensor temp_storage;
  temp_storage.mutable_data<uint8_t>(ctx.GetPlace(), temp_storage_bytes);

  if (descending) {
    err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        temp_storage.data<uint8_t>(), temp_storage_bytes, inp, sorted_out_ptr,
        input_indices.data<IndType>(), sorted_indices_ptr, num_cols * num_rows,
        num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
        cu_stream);
  } else {
    err = hipcub::DeviceSegmentedRadixSort::SortPairs(
        temp_storage.data<uint8_t>(), temp_storage_bytes, inp, sorted_out_ptr,
        input_indices.data<IndType>(), sorted_indices_ptr, num_cols * num_rows,
        num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
        cu_stream);
  }

  PADDLE_ENFORCE_CUDA_SUCCESS(err);
}

template <typename T, typename IndType>
void ArgFullAssign(const platform::CUDADeviceContext& ctx, const Tensor* dO,
                   const Tensor* indices, Tensor* dX, const IndType num_rows,
                   const IndType num_cols) {
  auto cu_stream = ctx.stream();

  auto ComputeBlockSize = [](IndType col) {
    if (col > 512)
      return 1024;
    else if (col > 256 && col <= 512)
      return 512;
    else if (col > 128 && col <= 256)
      return 256;
    else if (col > 64 && col <= 128)
      return 128;
    else
      return 64;
  };

  int block_size = ComputeBlockSize(num_cols);

  int maxGridDimX = ctx.GetCUDAMaxGridDimSize().x;
  // actually, int num_rows < max_grid_size
  int grid_size = num_rows < maxGridDimX ? num_rows : maxGridDimX;
  FillGrad<<<grid_size, block_size, 0, cu_stream>>>(
      dO->data<T>(), indices->data<IndType>(), dX->data<T>(), num_rows,
      num_cols);
}

template <typename T>
void ArgFlattenAssign(const platform::CUDADeviceContext& ctx, const Tensor* dO,
                      const Tensor* indices, int64_t size, Tensor* dX) {
  auto cu_stream = ctx.stream();

  const int64_t block_size =
      std::min(size, static_cast<int64_t>(ctx.GetMaxThreadsPerBlock()));
  int64_t max_threads = ctx.GetMaxPhysicalThreadCount();
  const int64_t max_blocks =
      std::max(((max_threads - 1) / block_size + 1), static_cast<int64_t>(1));
  const int64_t grid_size =
      std::min(max_blocks, (size + block_size - 1) / block_size);

  FillFlattenGrad<<<grid_size, block_size, 0, cu_stream>>>(
      dO->data<T>(), indices->data<int64_t>(), size, dX->data<T>());
}

template <typename DeviceContext, typename T>
class ArgsortOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");
    auto* indices = ctx.Output<Tensor>("Indices");
    int axis = ctx.Attr<int>("axis");
    bool descending = ctx.Attr<bool>("descending");

    auto in_dims = input->dims();
    axis = (axis < 0) ? (in_dims.size() + axis) : axis;

    const T* in_data = input->data<T>();
    auto size = input->numel();
    T* out_data = output->mutable_data<T>(ctx.GetPlace());
    int64_t* ids_data = indices->mutable_data<int64_t>(ctx.GetPlace());

    // Use thrust for parallel acceleration when the input size is equal to the
    // length of the ‘axis’ dimension.
    // Compared to the following 'Special case for full sort', ascending sort is
    // 34 times faster and descending sort is 31 times faster.
    if (size == in_dims[axis]) {
      thrust::sequence(thrust::device, ids_data, ids_data + size);
      thrust::copy(thrust::device, in_data, in_data + size, out_data);
      thrust::sort_by_key(thrust::device, out_data, out_data + size, ids_data);
      if (descending) {
        thrust::reverse(thrust::device, out_data, out_data + size);
        thrust::reverse(thrust::device, ids_data, ids_data + size);
      }
      return;
    }

    // Special case for full sort, speedup ~190x.
    if (axis == -1 || axis + 1 == in_dims.size()) {
      const int64_t input_height = framework::product(
          framework::slice_ddim(in_dims, 0, in_dims.size() - 1));
      const int64_t input_width = in_dims[in_dims.size() - 1];
      const auto& dev_ctx = ctx.cuda_device_context();
      ArgFullSort<T, int64_t>(dev_ctx, input, output, indices, input_height,
                              input_width, descending);
    } else {
      // if not full sort, do transpose first
      std::vector<int> trans;
      for (int i = 0; i < axis; i++) {
        trans.push_back(i);
      }
      trans.push_back(in_dims.size() - 1);
      for (int i = axis + 1; i < in_dims.size() - 1; i++) {
        trans.push_back(i);
      }
      trans.push_back(axis);
      framework::DDim trans_dims(in_dims);
      for (int i = 0; i < trans.size(); i++) {
        trans_dims[i] = in_dims[trans[i]];
      }

      Tensor trans_inp;
      T* trans_inp_data = trans_inp.mutable_data<T>(trans_dims, ctx.GetPlace());
      int ndims = trans.size();
      const auto& dev_ctx = ctx.cuda_device_context();
      // Do transpose
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, *input,
                                                   &trans_inp, trans);

      const int64_t input_height = framework::product(
          framework::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
      const int64_t input_width = trans_dims[trans_dims.size() - 1];

      Tensor tmp_out;
      tmp_out.mutable_data<T>(trans_dims, ctx.GetPlace());
      T* out_data = output->mutable_data<T>(ctx.GetPlace());

      Tensor tmp_indices;
      // temp indices for sorting
      tmp_indices.mutable_data<int64_t>(trans_dims, ctx.GetPlace());
      indices->mutable_data<int64_t>(ctx.GetPlace());

      ArgFullSort<T, int64_t>(dev_ctx, &trans_inp, &tmp_out, &tmp_indices,
                              input_height, input_width, descending);

      TransCompute<platform::CUDADeviceContext, int64_t>(
          ndims, dev_ctx, tmp_indices, indices, trans);
      // transpose back
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, tmp_out,
                                                   output, trans);
      return;
    }
  }
};

template <typename T>
class ArgsortGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* indices = ctx.Input<Tensor>("Indices");
    auto* dX = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* dO = ctx.Input<Tensor>(framework::GradVarName("Out"));
    int axis = ctx.Attr<int>("axis");

    dX->mutable_data<T>(ctx.GetPlace());
    if (dO->numel() == 0) return;

    auto in_dims = dX->dims();
    axis = (axis < 0) ? (in_dims.size() + axis) : axis;

    int64_t size = dX->numel();
    const auto& dev_ctx = ctx.cuda_device_context();

    // Parallel acceleration when the input size is equal to the length of the
    // ‘axis’ dimension.
    // Compared to 'special case for full sort' below, the gradient calculation
    // is 10 times faster.
    if (size == in_dims[axis]) {
      ArgFlattenAssign<T>(dev_ctx, dO, indices, size, dX);
      return;
    }

    // Special case for full sort, speedup ~190x.
    if (axis == -1 || axis + 1 == in_dims.size()) {
      const int64_t input_height = framework::product(
          framework::slice_ddim(in_dims, 0, in_dims.size() - 1));
      const int64_t input_width = in_dims[in_dims.size() - 1];
      ArgFullAssign<T, int64_t>(dev_ctx, dO, indices, dX, input_height,
                                input_width);
    } else {
      // if not full sort, do transpose first
      std::vector<int> trans;
      for (int i = 0; i < axis; i++) {
        trans.push_back(i);
      }
      trans.push_back(in_dims.size() - 1);
      for (int i = axis + 1; i < in_dims.size() - 1; i++) {
        trans.push_back(i);
      }
      trans.push_back(axis);
      framework::DDim trans_dims(in_dims);
      for (int i = 0; i < trans.size(); i++) {
        trans_dims[i] = in_dims[trans[i]];
      }

      Tensor trans_dO;
      trans_dO.mutable_data<T>(trans_dims, ctx.GetPlace());
      Tensor trans_ind;
      trans_ind.mutable_data<int64_t>(trans_dims, ctx.GetPlace());
      int ndims = trans.size();
      // Do transpose
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, *dO,
                                                   &trans_dO, trans);
      TransCompute<platform::CUDADeviceContext, int64_t>(
          ndims, dev_ctx, *indices, &trans_ind, trans);

      const int64_t input_height = framework::product(
          framework::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
      const int64_t input_width = trans_dims[trans_dims.size() - 1];

      Tensor tmp_out;
      tmp_out.mutable_data<T>(trans_dims, ctx.GetPlace());

      ArgFullAssign<T, int64_t>(dev_ctx, &trans_dO, &trans_ind, &tmp_out,
                                input_height, input_width);

      // transpose back
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, tmp_out, dX,
                                                   trans);
      return;
    }
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    argsort,
    paddle::operators::ArgsortOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                           float>,
    paddle::operators::ArgsortOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                           double>,
    paddle::operators::ArgsortOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                           int>,
    paddle::operators::ArgsortOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                           int64_t>,
    paddle::operators::ArgsortOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                           paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    argsort_grad, paddle::operators::ArgsortGradOpCUDAKernel<float>,
    paddle::operators::ArgsortGradOpCUDAKernel<double>,
    paddle::operators::ArgsortGradOpCUDAKernel<int>,
    paddle::operators::ArgsortGradOpCUDAKernel<int64_t>,
    paddle::operators::ArgsortGradOpCUDAKernel<paddle::platform::float16>);
