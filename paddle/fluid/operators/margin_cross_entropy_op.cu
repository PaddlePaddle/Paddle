#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// old op include, fluid should be removed
#ifdef PADDLE_WITH_HIP
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#else
#include <hipcub/hipcub.hpp>
#endif

#include <vector>
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/impl/softmax_kernel_impl.h"
#include "paddle/phi/kernels/margin_cross_entropy_grad_kernel.h"

#include "paddle/phi/common/memory_utils.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/process_group.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif
#include "paddle/phi/backends/gpu/gpu_context.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename Context>
void GetClassInterval(const gpuStream_t& stream,
                      const phi::Place& place,
                      const Context& dev_ctx,
                      const int rid,
                      const int rank,
                      const int nranks,
                      const int D,
                      DenseTensor* class_interval) {
  std::vector<int> shard_dim_vec(nranks + 1, 0);
  shard_dim_vec[rank + 1] = D;
  if (nranks <= 1) {
    phi::TensorFromVector(shard_dim_vec, dev_ctx, class_interval);
    return;
  }

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  DenseTensor num_classes_per_device;
  phi::TensorFromVector(shard_dim_vec, dev_ctx, &num_classes_per_device);
  int* num_classes_per_device_ptr = num_classes_per_device.data<int>();

  auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();
  if (map->has(rid)) {
    // Use ProcessGroup
    paddle::distributed::ProcessGroup* pg = map->get(rid);
    std::vector<phi::DenseTensor> in_tensor;
    std::vector<phi::DenseTensor> out_tensor;
    in_tensor.push_back(num_classes_per_device);
    out_tensor.push_back(num_classes_per_device);

    paddle::distributed::AllreduceOptions opts;
    opts.reduce_op = paddle::distributed::ReduceOp::SUM;
    auto task = pg->AllReduce(in_tensor, out_tensor, opts);
    task->Wait();
  } else {
    const auto& comm =
        paddle::platform::NCCLCommContext::Instance().Get(rid, place);
    // use global calculate stream
    const auto calcu_stream =
        static_cast<GPUContext*>(phi::DeviceContextPool::Instance().Get(place))
            ->stream();

    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::ncclAllReduce(
        num_classes_per_device_ptr,
        num_classes_per_device_ptr,
        num_classes_per_device.numel(),
        phi::ToNCCLDataType(num_classes_per_device.dtype()),
        ncclSum,
        comm->comm(),
        calcu_stream));
  }

  class_interval->Resize({nranks + 1});
  auto class_interval_ptr = dev_ctx.template Alloc<int>(class_interval);
  size_t cub_temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum<int*, int*>(
      nullptr, cub_temp_storage_bytes, nullptr, nullptr, nranks + 1, stream);
  auto cub_temp_storage =
      phi::memory_utils::Alloc(place, cub_temp_storage_bytes);
  hipcub::DeviceScan::InclusiveSum<int*, int*>(cub_temp_storage->ptr(),
                                            cub_temp_storage_bytes,
                                            num_classes_per_device_ptr,
                                            class_interval_ptr,
                                            nranks + 1,
                                            stream);
  return;
#endif
}

template <typename T, typename IndexT>
__global__ void AddMarginToPositiveLogitsKernel(T* logit,
                                                const IndexT* label,
                                                const float margin1,
                                                const float margin2,
                                                const float margin3,
                                                const int rank,
                                                const int nranks,
                                                const int64_t N,
                                                const int64_t D,
                                                const int* class_interval_ptr) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  int start_index = class_interval_ptr[rank];
  int end_index = class_interval_ptr[rank + 1];
  int num_classes = class_interval_ptr[nranks];
  CUDA_KERNEL_LOOP(i, N) {
    auto real_label = label[i];
    PADDLE_ENFORCE((real_label < num_classes) && (real_label >= 0),
                   "The index is out of bounds, "
                   "please check whether the value of label and "
                   "input meet the number of class. It should "
                   "be less than [%d], but received [%d]",
                   num_classes,
                   real_label);

    if (real_label >= start_index && real_label < end_index) {
      int64_t offset = i * D + real_label - start_index;
      if (fabs(margin1 - 1.0) > 1e-8 || fabs(margin2) > 1e-8) {
        MPType x = static_cast<MPType>(logit[offset]);
        MPType theta = acos(x);
        if (fabs(margin1 - 1.0) > 1e-8) {
          theta *= static_cast<MPType>(margin1);
        }
        if (fabs(margin2) > 1e-8) {
          theta += static_cast<MPType>(margin2);
        }
        logit[offset] = static_cast<T>(cos(theta));
      }
      if (fabs(margin3) > 1e-8) {
        MPType y = static_cast<MPType>(logit[offset]);
        y -= static_cast<MPType>(margin3);
        logit[offset] = static_cast<T>(y);
      }
    }
  }
}

template <typename T>
__global__ void ScaleLogitKernel(T* logits,
                                 const float scale,
                                 const int64_t N,
                                 const int64_t D) {
  CUDA_KERNEL_LOOP(i, N * D) { logits[i] *= static_cast<T>(scale); }
}

template <typename T>
__global__ void LogitsMinusMaxKernel(T* logits,
                                     const T* logits_max_per_row,
                                     const int64_t N,
                                     const int64_t D) {
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    logits[i] -= logits_max_per_row[row];
  }
}

template <typename T>
__global__ void LogitsMinusLogSumKernel(T* logits,
                                        const T* logits_sum_per_row,
                                        const int64_t N,
                                        const int64_t D) {
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    logits[i] -= phi::kps::details::Log(logits_sum_per_row[row]);
  }
}

template <typename T, typename IndexT>
__global__ void HardLabelSoftmaxWithCrossEntropyKernel(
    T* loss,
    T* log_softmax,
    const IndexT* labels,
    const int rank,
    const int64_t N,
    const int64_t D,
    const int* class_interval_ptr) {
  int start_index = class_interval_ptr[rank];
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    auto col = i % D;
    if ((col + start_index) == labels[row]) {
      auto softmax = log_softmax[i];
      loss[row] = -softmax;
      log_softmax[i] = phi::kps::details::Exp(softmax);
    } else {
      log_softmax[i] = phi::kps::details::Exp(log_softmax[i]);
    }
  }
}

template <typename T, typename Context>
void MarginCrossEntropyKernel(const Context& dev_ctx,
                              const DenseTensor& logits,
                              const DenseTensor& labels,
                              bool return_softmax,
                              int ring_id,
                              int rank,
                              int nranks,
                              float margin1,
                              float margin2,
                              float margin3,
                              float scale,
                              DenseTensor* softmax,
                              DenseTensor* loss) {
  const auto& place = dev_ctx.GetPlace();  // old code

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  paddle::platform::NCCLComm* comm;
  paddle::distributed::ProcessGroup* pg = nullptr;
  gpuStream_t stream;
  if (nranks > 1) {
    auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();
    if (map->has(ring_id)) {
      // Use ProcessGroup
      pg = map->get(ring_id);
    } else {
      comm = paddle::platform::NCCLCommContext::Instance().Get(ring_id, place);

      // use global calculate stream
      stream = static_cast<GPUContext*>(
                   phi::DeviceContextPool::Instance().Get(place))
                   ->stream();
    }
  }
#endif

  // allocate memory on device.
  T* softmax_ptr = dev_ctx.template Alloc<T>(softmax);
  T* loss_ptr = dev_ctx.template Alloc<T>(loss);

  const auto& logits_dims = logits.dims();
  const auto& labels_dims = labels.dims();

  const int axis = logits_dims.size() - 1;
  const int N = phi::funcs::SizeToAxis(axis, logits_dims);
  const int D = phi::funcs::SizeFromAxis(axis, logits_dims);

  int blocks = NumBlocks(N);
  int threads = kNumCUDAThreads;
  const auto& label_type = labels.dtype();

  // copy logits to softmax variable since we can't modify logits,
  // and it also be used when calculate grad
  phi::Copy<Context>(dev_ctx, logits, dev_ctx.GetPlace(), true, softmax);

  DenseTensor softmax_2d;
  softmax_2d.ShareDataWith(*softmax).Resize({N, D});
  T* logits_ptr = softmax_2d.data<T>();

  DenseTensor class_interval;
  GetClassInterval<T, Context>(dev_ctx.stream(),
                               dev_ctx.GetPlace(),
                               dev_ctx,
                               ring_id,
                               rank,
                               nranks,
                               D,
                               &class_interval);

  // step 1, preprocess logits
  // add margin for positive elements
  // theta = acos(x_i)
  // (cos(m1 * theta + m2) - m3)
  // save match_logits, used for gradient computation.
  if (label_type == phi::DataType::INT32) {
    typedef int32_t LabelT;
    AddMarginToPositiveLogitsKernel<T>
        <<<NumBlocks(N), threads, 0, dev_ctx.stream()>>>(
            logits_ptr,
            labels.data<LabelT>(),
            margin1,
            margin2,
            margin3,
            rank,
            nranks,
            N,
            D,
            class_interval.data<int>());
  } else if (label_type == phi::DataType::INT64) {
    typedef int64_t LabelT;
    AddMarginToPositiveLogitsKernel<T>
        <<<NumBlocks(N), threads, 0, dev_ctx.stream()>>>(
            logits_ptr,
            labels.data<LabelT>(),
            margin1,
            margin2,
            margin3,
            rank,
            nranks,
            N,
            D,
            class_interval.data<int>());
  } else {
    PADDLE_THROW(errors::Unimplemented(
        "margin_cross_entropy label type noly support int32 and int64, "
        "but got %s",
        label_type));
  }

  // scale by s
  ScaleLogitKernel<T><<<NumBlocks(N * D), threads, 0, dev_ctx.stream()>>>(
      logits_ptr, scale, N, D);

  // step 2, obtain logit_max
  DenseTensor logits_max;
  logits_max.Resize({N, 1});
  dev_ctx.template Alloc<T>(&logits_max);
  T* logits_max_buff = dev_ctx.template Alloc<T>(&logits_max);

  phi::funcs::
      ReduceKernel<T, T, phi::kps::MaxFunctor, phi::kps::IdentityFunctor<T>>(
          static_cast<const phi::GPUContext&>(dev_ctx),
          softmax_2d,
          &logits_max,
          phi::kps::IdentityFunctor<T>(),
          {1});

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    if (pg) {
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(logits_max);
      out_tensor.push_back(logits_max);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::MAX;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::ncclAllReduce(logits_max_buff,
                                      logits_max_buff,
                                      logits_max.numel(),
                                      phi::ToNCCLDataType(logits_max.dtype()),
                                      ncclMax,
                                      comm->comm(),
                                      stream));
    }
  }
#endif

  // step 3, logit - logit_max
  LogitsMinusMaxKernel<T><<<NumBlocks(N * D), threads, 0, dev_ctx.stream()>>>(
      logits_ptr, logits_max_buff, N, D);

  // step 4, sum(exp(logit - logit_max))
  DenseTensor sum_exp_logits;
  sum_exp_logits.Resize({N, 1});
  dev_ctx.template Alloc<T>(&sum_exp_logits);
  T* sum_exp_logits_buff = dev_ctx.template Alloc<T>(&sum_exp_logits);
  phi::funcs::ReduceKernel<T, T, phi::kps::AddFunctor, phi::kps::ExpFunctor<T>>(
      static_cast<const phi::GPUContext&>(dev_ctx),
      softmax_2d,
      &sum_exp_logits,
      phi::kps::ExpFunctor<T>(),
      {1});

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    if (pg) {
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(sum_exp_logits);
      out_tensor.push_back(sum_exp_logits);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::SUM;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::ncclAllReduce(
          sum_exp_logits_buff,
          sum_exp_logits_buff,
          sum_exp_logits.numel(),
          phi::ToNCCLDataType(sum_exp_logits.dtype()),
          ncclSum,
          comm->comm(),
          stream));
    }
  }
#endif

  // step 5, (logit - logit_max) - log(sum(exp(logit - logit_max)))
  LogitsMinusLogSumKernel<T>
      <<<NumBlocks(N * D), threads, 0, dev_ctx.stream()>>>(
          logits_ptr, sum_exp_logits_buff, N, D);

  // step 6, prob = exp((logit - logit_max) - log(sum(exp(logit -
  // logit_max))))
  // loss = -((logit_i - logit_max) - log(sum(exp(logit - logit_max))))

  phi::funcs::SetConstant<Context, T> functor;
  functor(dev_ctx, loss, static_cast<T>(0.0));
  if (label_type == phi::DataType::INT32) {
    typedef int32_t LabelT;
    HardLabelSoftmaxWithCrossEntropyKernel<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_ptr,
                                                   logits_ptr,
                                                   labels.data<LabelT>(),
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  } else if (label_type == phi::DataType::INT64) {
    typedef int64_t LabelT;
    HardLabelSoftmaxWithCrossEntropyKernel<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_ptr,
                                                   logits_ptr,
                                                   labels.data<LabelT>(),
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  }

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    if (pg) {
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(*loss);
      out_tensor.push_back(*loss);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::SUM;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::ncclAllReduce(loss_ptr,
                                      loss_ptr,
                                      loss->numel(),
                                      phi::ToNCCLDataType(loss->dtype()),
                                      ncclSum,
                                      comm->comm(),
                                      stream));
    }
  }
#endif
}

template <typename T, typename IndexT>
__global__ void CalculateGrad(T* logits_grad,
                              const T* loss_grad,
                              const T* logits,
                              const IndexT* label,
                              const float margin1,
                              const float margin2,
                              const float scale,
                              const int rank,
                              const int64_t N,
                              const int64_t D,
                              const int* class_interval_ptr) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  int start_index = class_interval_ptr[rank];
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    auto col = i % D;
    if ((col + start_index) == label[row]) {
      logits_grad[i] = (logits_grad[i] - static_cast<T>(1.0)) * loss_grad[row];
      if (fabs(margin1 - 1.0) > 1e-8 || fabs(margin2) > 1e-8) {
        MPType dout = static_cast<MPType>(logits_grad[i]);
        MPType one = static_cast<MPType>(1.0f);
        MPType x = static_cast<MPType>(logits[i]);
        MPType m1 = static_cast<MPType>(margin1);
        MPType m2 = static_cast<MPType>(margin2);

        MPType d = m1 * sin(m1 * acos(x) + m2) / sqrt(one - x * x);
        logits_grad[i] = static_cast<T>(dout * d);
      }
    } else {
      logits_grad[i] *= loss_grad[row];
    }
    if (fabs(scale - 1.0) > 1e-8) {
      logits_grad[i] *= static_cast<T>(scale);
    }
  }
}

template <typename T, typename Context>
void MarginCrossEntropyGradKernel(const Context& dev_ctx,
                                  const DenseTensor& logits,
                                  const DenseTensor& label,
                                  const DenseTensor& softmax,
                                  const DenseTensor& loss_grad,
                                  bool return_softmax,
                                  int ring_id,
                                  int rank,
                                  int nranks,
                                  float margin1,
                                  float margin2,
                                  float margin3,
                                  float scale,
                                  DenseTensor* logits_grad) {
  const auto softmax_dims = softmax.dims();
  const int axis = softmax_dims.size() - 1;
  const int N = phi::funcs::SizeToAxis(axis, softmax_dims);
  const int D = phi::funcs::SizeFromAxis(axis, softmax_dims);

  if (return_softmax) {
    phi::Copy<Context>(
        dev_ctx, softmax, dev_ctx.GetPlace(), false, logits_grad);
  } else {
    logits_grad->ShareDataWith(softmax);
  }

  int blocks = NumBlocks(N * D);
  int threads = kNumCUDAThreads;
  const auto& label_type = label.dtype();

  DenseTensor class_interval;
  GetClassInterval<T, Context>(dev_ctx.stream(),
                               dev_ctx.GetPlace(),
                               dev_ctx,
                               ring_id,
                               rank,
                               nranks,
                               D,
                               &class_interval);

  if (label_type == phi::DataType::INT32) {
    typedef int32_t LabelT;
    CalculateGrad<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logits_grad->data<T>(),
                                                   loss_grad.data<T>(),
                                                   logits.data<T>(),
                                                   label.data<LabelT>(),
                                                   margin1,
                                                   margin2,
                                                   scale,
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  } else if (label_type == phi::DataType::INT64) {
    typedef int64_t LabelT;
    CalculateGrad<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logits_grad->data<T>(),
                                                   loss_grad.data<T>(),
                                                   logits.data<T>(),
                                                   label.data<LabelT>(),
                                                   margin1,
                                                   margin2,
                                                   scale,
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(margin_cross_entropy,
                   GPU,
                   ALL_LAYOUT,
                   phi::MarginCrossEntropyKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(margin_cross_entropy_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::MarginCrossEntropyGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
