#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/cum_op.h"
#include "paddle/fluid/platform/gpu_launch_param_config.h"

using Tensor = paddle::framework::Tensor;
using LoDTensor = paddle::framework::LoDTensor;
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFILICT_FREE_OFFSET(n) \
  ((n >> LOG_NUM_BANKS) + (n) >> (2 * LOG_NUM_BANKS))
#define CUDA_ERROR(err, msg)                                            \
  {                                                                     \
    if (err != hipSuccess) {                                           \
      printf("%s: %s in %s at line %d\n", msg, hipGetErrorString(err), \
             __FILE__, __LINE__);                                       \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  }
namespace paddle {
namespace operators {

template <typename T>

// inclusive scan
__global__ void OuterScan(const T* in, T* out, unsigned inner_dim_size,
                          unsigned outer_dim_size, unsigned scan_dim_size,
                          bool exclusive) {
  int id = blockIdx.y * blockDim.x + threadIdx.x;

  for (unsigned outer_index = blockIdx.x; outer_index < outer_dim_size;
       outer_index += gridDim.x) {
    for (unsigned inner_index = blockIdx.y * blockDim.x + threadIdx.x;
         inner_index < inner_dim_size; inner_index += gridDim.y * blockDim.x) {
      int src_index =
          outer_index * scan_dim_size * inner_dim_size + inner_index;
      int dst_index =
          outer_index * scan_dim_size * inner_dim_size + inner_index;
      T acc = 0;
      for (unsigned scan_index = 0; scan_index < scan_dim_size; ++scan_index) {
        acc = in[src_index] + acc;
        out[dst_index] = acc;
        src_index += inner_dim_size;
        dst_index += inner_dim_size;
      }
    }
  }
}

// exclusive scan
template <typename T>
__global__ void BlellochScan(const T* in, T* out, unsigned inner_dim_size,
                             unsigned outer_dim_size, unsigned scan_dim_size,
                             int size) {
  // https://stackoverflow.com/questions/27570552/templated-cuda-kernel-with-dynamic-shared-memory
  extern __shared__ __align__(sizeof(T)) unsigned char raw_tmp[];
  T* share_tmp = reinterpret_cast<T*>(raw_tmp);

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int thread_idx = threadIdx.x;
  int offset = 1;
  int element_size = inner_dim_size;
  int active_thread_size = inner_dim_size / 2;
  if (idx > active_thread_size) return;

  for (; idx < active_thread_size; idx += blockDim.x * gridDim.x) {
    for (size_t i = 0; i < outer_dim_size; i++) {
      // load data to share memory
      share_tmp[2 * thread_idx] = in[(2 * idx) + inner_dim_size * i];
      if ((2 * thread_idx + 1) < element_size) {
        share_tmp[2 * thread_idx + 1] = in[(2 * idx + 1) + inner_dim_size * i];
      }
      __syncthreads();

      // parallel reduction(up-sweep)
      for (int s = element_size >> 1; s > 0; s >>= 1) {
        if (thread_idx < s && idx < active_thread_size) {
          int ai = offset * (2 * thread_idx + 1) - 1;
          int bi = offset * (2 * thread_idx + 2) - 1;
          share_tmp[bi] += share_tmp[ai];
        }
        offset *= 2;
        __syncthreads();
      }
      // set the last element to be zero
      if (thread_idx == 0) share_tmp[element_size - 1] = 0;
      __syncthreads();

      // Down-sweep
      for (int s = 1; s < element_size; s <<= 1) {
        offset >>= 1;
        if (thread_idx < s && idx < active_thread_size) {
          int ai = offset * (2 * thread_idx + 1) - 1;
          int bi = offset * (2 * thread_idx + 2) - 1;
          T tmp = share_tmp[ai];
          share_tmp[ai] = share_tmp[bi];
          share_tmp[bi] += tmp;
        }
        __syncthreads();
      }

      // write back to memory
      if (thread_idx < active_thread_size) {
        out[(2 * idx) + inner_dim_size * i] = share_tmp[2 * thread_idx];
        out[(2 * idx + 1) + inner_dim_size * i] = share_tmp[2 * thread_idx + 1];
      }
    }
  }
}

template <typename DeviceContext, typename T>
class CumCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<framework::Tensor>("X");
    auto* out = context.Output<framework::Tensor>("Out");

    int axis = context.Attr<int>("axis");
    bool exclusive = context.Attr<bool>("exclusive");
    bool reverse = context.Attr<bool>("reverse");
    auto in_dims = in->dims();
    auto size = in->numel();

    if (axis == -1) {
      axis = in_dims.size() - 1;
    }
    PADDLE_ENFORCE_LT(
        axis, in_dims.size(),
        platform::errors::InvalidArgument("axis(%d) should be less than the "
                                          "dimension(%d) of the input tensor.",
                                          axis, in_dims.size()));

    unsigned scan_dim_size = in_dims[axis];
    unsigned outer_dim_size = 1;
    unsigned inner_dim_size = 1;
    // treat all dim index < axis as outer_dim_size
    for (size_t i = 0; i < axis; i++) {
      outer_dim_size *= in_dims[i];
    }
    // treat all dim index > axis as innner_dim_size
    for (size_t i = axis + 1; i < in_dims.size(); i++) {
      inner_dim_size *= in_dims[i];
    }

    T* out_data = out->mutable_data<T>(context.GetPlace());
    const T* in_data = in->data<T>();

    auto& dev_ctx = context.template device_context<DeviceContext>();
    bool optimize_condition = false;
    if (optimize_condition) {
      int mem_per_block = size * sizeof(T);
      dim3 block(1024);
      dim3 grid((size + block.x - 1) / block.x);

    } else {
      dim3 block(std::min(512u, inner_dim_size));
      dim3 grid(outer_dim_size, (inner_dim_size + block.x - 1) / block.x);
      OuterScan<T><<<block, grid, 0, dev_ctx.stream()>>>(
          in_data, out_data, inner_dim_size, outer_dim_size, scan_dim_size,
          exclusive);
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    cumsum, ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
