#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/cum_op.h"
#include "paddle/fluid/platform/gpu_launch_param_config.h"

using Tensor = paddle::framework::Tensor;
using LoDTensor = paddle::framework::LoDTensor;

namespace paddle {
namespace operators {

template <typename T>
__global__ void OuterScan(const T* in, T* out, int inner_dim_size,
                          int outer_dim_size, int scan_dim_size, bool exclusive,
                          bool reverse) {
  int id = blockIdx.y * blockDim.x + threadIdx.x;

  for (int outer_index = blockIdx.x; outer_index < outer_dim_size;
       outer_index += gridDim.x) {
    for (int inner_index = blockIdx.y * blockDim.x + threadIdx.x;
         inner_index < inner_dim_size; inner_index += gridDim.y * blockDim.x) {
      int scan_index_init = 0;
      int forward_direction = 1;
      int src_index =
          outer_index * scan_dim_size * inner_dim_size + inner_index;
      int dst_index =
          outer_index * scan_dim_size * inner_dim_size + inner_index;
      if (reverse) {
        src_index = src_index + (scan_dim_size - 1) * inner_dim_size;
        dst_index = dst_index + (scan_dim_size - 1) * inner_dim_size;
        forward_direction = -1;
      }
      if (exclusive) {
        scan_index_init = 1;
        out[dst_index] = 0;
        dst_index = dst_index + (forward_direction * inner_dim_size);
      }
      T acc = 0;

      for (int scan_index = scan_index_init; scan_index < scan_dim_size;
           ++scan_index) {
        acc = in[src_index] + acc;
        out[dst_index] = acc;
        src_index += (forward_direction * inner_dim_size);
        dst_index += (forward_direction * inner_dim_size);
      }
    }
  }
}

// inclusive scan
template <typename T, int num_threads_x, int num_threads_y>
__global__ void InnerMostDimInclusiveScan(const T* in, T* out,
                                          int inner_dim_size,
                                          int outer_dim_size, int scan_dim_size,
                                          bool reverse) {
  __shared__ T share_data[num_threads_y][num_threads_x * 2];
  T* share_row = share_data[threadIdx.y];
  int forward_direction = 1;
  if (reverse) forward_direction = -1;

  for (int block_row = blockIdx.x * blockDim.y; block_row < outer_dim_size;
       block_row += blockDim.y * gridDim.x) {
    int row = block_row + threadIdx.y;
    T acc = 0;
    const T* row_src = in + row * scan_dim_size;
    T* row_dst = out + row * scan_dim_size;
    int block_col = 0;
    bool loop_condition = (block_col < scan_dim_size);
    if (reverse) {
      loop_condition = (block_col >= 0);
      block_col = scan_dim_size - 1;
    }
    while (loop_condition) {
      // Load data into share memory(two value per thread)
      int col1 = block_col + threadIdx.x * forward_direction;
      int col2 = block_col + (num_threads_x + threadIdx.x) * forward_direction;
      if (row < outer_dim_size) {
        if (col1 < scan_dim_size && col1 >= 0) {
          share_row[threadIdx.x] = row_src[col1];
        } else {
          share_row[threadIdx.x] = 0;
        }

        if (col2 < scan_dim_size && col2 >= 0) {
          share_row[num_threads_x + threadIdx.x] = row_src[col2];
        } else {
          share_row[num_threads_x + threadIdx.x] = 0;
        }

        // Add the previous block acc to the result
        if (threadIdx.x == 0) {
          share_row[0] = share_row[0] + acc;
        }
      }
      __syncthreads();

      // Up-Sweep
      for (unsigned s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < outer_dim_size && threadIdx.x < s) {
          unsigned offset = (2 * threadIdx.x + 1) * d - 1;
          share_row[offset + d] = share_row[offset] + share_row[offset + d];
        }
        __syncthreads();
      }
      // Down-Sweep
      for (unsigned s = 2, d = blockDim.x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < outer_dim_size && threadIdx.x < s - 1) {
          unsigned offset = 2 * (threadIdx.x + 1) * d - 1;
          share_row[offset + d] = share_row[offset] + share_row[offset + d];
        }
        __syncthreads();
      }

      // Write to the output
      if (row < outer_dim_size) {
        if (col1 < scan_dim_size && col1 >= 0)
          row_dst[col1] = share_row[threadIdx.x];
        if (col2 < scan_dim_size && col2 >= 0)
          row_dst[col2] = share_row[num_threads_x + threadIdx.x];
      }
      acc = share_row[2 * num_threads_x - 1];
      __syncthreads();
      block_col += 2 * num_threads_x * forward_direction;
      if (reverse)
        loop_condition = (block_col >= 0);
      else
        loop_condition = (block_col < scan_dim_size);
    }
  }
}

// exclusive block scan and store block sum for large scan
template <typename T>
__global__ void InnerMostDimExclusiveScan(const T* in, T* out, T* sum_data,
                                          int inner_dim_size,
                                          int outer_dim_size, int scan_dim_size,
                                          int two_power, bool reverse) {
  // https://stackoverflow.com/questions/27570552/templated-cuda-kernel-with-dynamic-shared-memory
  extern __shared__ __align__(sizeof(T)) unsigned char raw_tmp[];
  T* share_tmp = reinterpret_cast<T*>(raw_tmp);
  int thread_id = threadIdx.x;
  int block_id = blockIdx.x;
  int block_scan_size = blockDim.x * 2;
  int remain = scan_dim_size % (2 * blockDim.x);
  if (block_id == gridDim.x - 1 && remain != 0) block_scan_size = remain;
  int col1 = thread_id;
  int col2 = thread_id + (block_scan_size) / 2;
  int index1 = blockIdx.y * (scan_dim_size) + block_id * blockDim.x * 2 + col1;
  int index2 = blockIdx.y * (scan_dim_size) + block_id * blockDim.x * 2 + col2;
  if (reverse) {
    index1 = blockIdx.y * (scan_dim_size) + scan_dim_size - 1 -
             (block_id * blockDim.x * 2 + col1);
    index2 = blockIdx.y * (scan_dim_size) + scan_dim_size - 1 -
             (block_id * blockDim.x * 2 + col2);
  }
  int sum_index = blockIdx.y * gridDim.x + block_id;
  if (thread_id < block_scan_size) {
    share_tmp[col1 + (col1 >> 5)] = in[index1];
    share_tmp[col2 + (col2 >> 5)] = in[index2];
  } else {
    share_tmp[col1 + (col1 >> 5)] = 0;
    share_tmp[col2 + (col2 >> 5)] = 0;
  }

  // Up-Sweep
  int offset = 1;
  for (int d = (two_power / 2); d > 0; d >>= 1) {
    __syncthreads();
    if (thread_id < d) {
      int tmp_index1 = offset * (2 * thread_id + 1) - 1;
      int tmp_index2 = offset * (2 * thread_id + 2) - 1;
      tmp_index1 = tmp_index1 + (tmp_index1 >> 5);
      tmp_index2 = tmp_index2 + (tmp_index2 >> 5);

      share_tmp[tmp_index2] += share_tmp[tmp_index1];
    }
    offset *= 2;
  }
  __syncthreads();

  if (thread_id == 0) {
    int tmp_index = (two_power - 1) + ((two_power - 1) >> 5);
    sum_data[sum_index] = share_tmp[tmp_index];
    share_tmp[tmp_index] = 0;
  }

  // Down Sweep
  for (int d = 1; d < two_power; d *= 2) {
    offset >>= 1;
    __syncthreads();
    if (thread_id < d) {
      int tmp_index1 = offset * (2 * thread_id + 1) - 1;
      int tmp_index2 = offset * (2 * thread_id + 2) - 1;
      tmp_index1 = tmp_index1 + (tmp_index1 >> 5);
      tmp_index2 = tmp_index2 + (tmp_index2 >> 5);

      T tmp = share_tmp[tmp_index1];
      share_tmp[tmp_index1] = share_tmp[tmp_index2];
      share_tmp[tmp_index2] += tmp;
    }
  }

  __syncthreads();

  if (col1 < block_scan_size) out[index1] = share_tmp[col1 + (col1 >> 5)];
  if (col2 < block_scan_size) out[index2] = share_tmp[col2 + (col2 >> 5)];
}

// for large scan_dim_size array we need to add for correct result
template <typename T>
__global__ void AddBlockScan(T* result, T* sum, int size, int scan_dim_size,
                             int sum_size, bool reverse) {
  int idx = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);
  int block_id_start = blockIdx.y * sum_size;
  int block_id_end = blockIdx.x + blockIdx.y * sum_size;
  int block_id = blockIdx.x;
  int thread_id = threadIdx.x;

  int col = block_id * blockDim.x + thread_id + size;
  int index = blockIdx.y * (scan_dim_size) + col;
  if (reverse) {
    index = blockIdx.y * (scan_dim_size) + scan_dim_size - 1 - col;
  }

  if (col >= scan_dim_size || col < 0) return;
  for (int i = block_id_start; i <= block_id_end; i++) {
    result[index] += sum[i];
  }
}

template <typename DeviceContext, typename T>
class CumCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<framework::Tensor>("X");
    auto* out = context.Output<framework::Tensor>("Out");

    int axis = context.Attr<int>("axis");
    bool exclusive = context.Attr<bool>("exclusive");
    bool reverse = context.Attr<bool>("reverse");
    auto in_dims = in->dims();
    auto size = in->numel();

    if (axis == -1) {
      axis = in_dims.size() - 1;
    }
    PADDLE_ENFORCE_LT(
        axis, in_dims.size(),
        platform::errors::InvalidArgument("axis(%d) should be less than the "
                                          "dimension(%d) of the input tensor.",
                                          axis, in_dims.size()));

    int scan_dim_size = in_dims[axis];
    bool optimize_condition = (axis == (in_dims.size() - 1)) ? true : false;
    int outer_dim_size = 1;
    int inner_dim_size = 1;
    // treat all dim index < axis as outer_dim_size
    for (size_t i = 0; i < axis; i++) {
      outer_dim_size *= in_dims[i];
    }
    // treat all dim index > axis as innner_dim_size
    for (size_t i = axis + 1; i < in_dims.size(); i++) {
      inner_dim_size *= in_dims[i];
    }

    T* out_data = out->mutable_data<T>(context.GetPlace());
    const T* in_data = in->data<T>();

    auto& dev_ctx = context.template device_context<DeviceContext>();
    if (optimize_condition) {
      auto nextPowerOfTwo = [](int x) -> int {
        int ret = 1;
        while (ret < x) ret = ret * 2;
        return ret;
      };
      if (exclusive) {
        int element_per_block = nextPowerOfTwo(scan_dim_size) / 2;
        if (element_per_block > 512 || element_per_block < 32) {
          element_per_block = 64;
        }
        int two_power = element_per_block * 2;
        dim3 block(element_per_block);
        dim3 grid(((scan_dim_size + 1) / 2 + block.x - 1) / block.x,
                  outer_dim_size);
        int offset_size = (element_per_block * 2) >> 5;
        int share_mem_size = (element_per_block * 2 + offset_size) * sizeof(T);
        Tensor scan_sum;
        paddle::framework::DDim dims{
            ((scan_dim_size + 1) / 2 + block.x - 1) / block.x, outer_dim_size};
        scan_sum.Resize(dims);
        T* sum_data = scan_sum.mutable_data<T>(context.GetPlace());
        InnerMostDimExclusiveScan<
            T><<<grid, block, share_mem_size, dev_ctx.stream()>>>(
            in_data, out_data, sum_data, inner_dim_size, outer_dim_size,
            scan_dim_size, two_power, reverse);
        // for large scan array we need to do add for correct result
        int element_size = element_per_block * 2;
        if (scan_dim_size > element_size) {
          dim3 sum_block(element_per_block * 2);
          dim3 sum_grid((scan_dim_size - element_size + block.x - 1) / block.x,
                        outer_dim_size);
          int sum_size = ((scan_dim_size + 1) / 2 + block.x - 1) / block.x;
          AddBlockScan<T><<<sum_grid, sum_block, 0, dev_ctx.stream()>>>(
              out_data, sum_data, element_size, scan_dim_size, sum_size,
              reverse);
        }

      } else {
        dim3 block(32, 16);
        dim3 grid((outer_dim_size + block.y - 1) / block.y);
        InnerMostDimInclusiveScan<T, 32,
                                  16><<<grid, block, 0, dev_ctx.stream()>>>(
            in_data, out_data, inner_dim_size, outer_dim_size, scan_dim_size,
            reverse);
      }
    } else {
      dim3 block(std::min(512, inner_dim_size));
      dim3 grid(outer_dim_size, (inner_dim_size + block.x - 1) / block.x);
      OuterScan<T><<<grid, block, 0, dev_ctx.stream()>>>(
          in_data, out_data, inner_dim_size, outer_dim_size, scan_dim_size,
          exclusive, reverse);
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    cumsum, ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::CumCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
