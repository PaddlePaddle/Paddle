#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fill_diagonal_tensor_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using CUDADeviceContext = paddle::platform::CUDADeviceContext;

template <typename T>
__global__ void fill_diagonal_tensor_kernel(int64_t size, T *out_data,
                                            const T *fill_data,
                                            int64_t *strides, int64_t *matdim,
                                            int64_t offset, int64_t fill_dims0,
                                            int64_t fill_dims1) {
  int64_t i = blockIdx.x;
  auto sumoff = matdim[i] + offset;
  for (int64_t j = threadIdx.x; j < fill_dims1; j += blockDim.x) {
    auto fill_index = j * (strides[1] + strides[0]) + sumoff;
    if (fill_index < size) {
      out_data[fill_index] = fill_data[i * fill_dims1 + j];
    }
  }
}

template <typename T>
__global__ void fill_grad_kernel(int64_t size, T *out_data, int64_t *strides,
                                 int64_t *matdim, int64_t offset,
                                 int64_t fill_dims0, int64_t fill_dims1) {
  int64_t i = blockIdx.x;
  auto sumoff = matdim[i] + offset;
  for (int64_t j = threadIdx.x; j < fill_dims1; j += blockDim.x) {
    auto fill_index = j * (strides[1] + strides[0]) + sumoff;
    if (fill_index < size) {
      out_data[fill_index] = T(0);
    }
  }
}

template <typename T>
class FillDiagonalTensorCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
#ifdef __HIPCC__
    const int64_t kMaxBlockDim = 256;
#else
    const int64_t kMaxBlockDim = 512;
#endif
    auto *out = ctx.Output<framework::Tensor>("Out");
    auto *srctensor = ctx.Input<framework::Tensor>("Y");
    auto dim1 = ctx.Attr<int>("dim1");
    auto dim2 = ctx.Attr<int>("dim2");
    auto offset = ctx.Attr<int64_t>("offset");

    auto *xin = ctx.Input<framework::Tensor>("X");
    framework::TensorCopy(*xin, ctx.GetPlace(), out);

    T *out_data = out->mutable_data<T>(ctx.GetPlace());
    const T *fill_data = srctensor->data<T>();

    auto out_dims = out->dims();
    auto matdims = srctensor->dims();
    auto fill_dims = phi::flatten_to_2d(matdims, matdims.size() - 1);

    int64_t new_dims[2];
    std::vector<int64_t> memory_block;
    memory_block.resize(2 + fill_dims[0]);
    int64_t *strides = &(memory_block[0]);
    int64_t *matdim = &(memory_block[2]);
    CalMatDims(out_dims, dim1, dim2, &offset, new_dims, strides, matdim);
    PADDLE_ENFORCE_EQ(
        new_dims[0], fill_dims[0],
        platform::errors::InvalidArgument("The dims should be %d x %d, but get "
                                          "%d x %d in fill tensor Y",
                                          new_dims[0], new_dims[1],
                                          fill_dims[0], fill_dims[1]));
    PADDLE_ENFORCE_EQ(
        new_dims[1], fill_dims[1],
        platform::errors::InvalidArgument("The dims should be %d x %d, but get "
                                          "%d x %d in fill tensor Y",
                                          new_dims[0], new_dims[1],
                                          fill_dims[0], fill_dims[1]));

    auto size = out->numel();

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    Tensor tensor_tmp;
    int64_t *memory_block_cu =
        tensor_tmp.mutable_data<int64_t>({2 + fill_dims[0]}, ctx.GetPlace());
    const auto gpu_place = ctx.GetPlace();
    memory::Copy(gpu_place, memory_block_cu, platform::CPUPlace(),
                 memory_block.data(), sizeof(int64_t) * (2 + fill_dims[0]),
                 stream);

    int64_t *strides_cu = &memory_block_cu[0], *matdim_cu = &memory_block_cu[2];

    auto kGridDim = new_dims[0];
    auto kBlockDim = std::min(int64_t(new_dims[1]), kMaxBlockDim);
    fill_diagonal_tensor_kernel<T><<<kGridDim, kBlockDim, 0, stream>>>(
        size, out_data, fill_data, strides_cu, matdim_cu, offset, fill_dims[0],
        fill_dims[1]);
  }
};

template <typename T>
class FillDiagonalTensorGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
#ifdef __HIPCC__
    const int64_t kMaxBlockDim = 256;
#else
    const int64_t kMaxBlockDim = 512;
#endif
    auto *dx = ctx.Output<framework::Tensor>(framework::GradVarName("X"));
    auto *dout = ctx.Input<framework::Tensor>(framework::GradVarName("Out"));

    auto dim1 = ctx.Attr<int>("dim1");
    auto dim2 = ctx.Attr<int>("dim2");
    auto offset = ctx.Attr<int64_t>("offset");
    auto matrows = 1;

    if (dx) {
      auto *data = dx->mutable_data<T>(ctx.GetPlace());
      auto dx_dims = dx->dims();
      framework::TensorCopy(*dout, ctx.GetPlace(), dx);

      for (int i = 0; i < dx_dims.size(); i++) {
        if (i != dim1 && i != dim2) {
          matrows *= dx_dims[i];
        }
      }

      int64_t new_dims[2];
      std::vector<int64_t> memory_block;
      memory_block.resize(2 + matrows);
      int64_t *strides = &memory_block[0];
      int64_t *matdim = &memory_block[2];
      CalMatDims(dx_dims, dim1, dim2, &offset, new_dims, strides, matdim);

      auto size = dx->numel();

      auto &dev_ctx =
          ctx.template device_context<platform::CUDADeviceContext>();
      auto stream = dev_ctx.stream();
      Tensor tensor_tmp;
      int64_t *memory_block_cu =
          tensor_tmp.mutable_data<int64_t>({2 + matrows}, ctx.GetPlace());
      const auto gpu_place = ctx.GetPlace();
      memory::Copy(gpu_place, memory_block_cu, platform::CPUPlace(),
                   memory_block.data(), sizeof(int64_t) * (2 + matrows),
                   stream);

      int64_t *strides_cu = &memory_block_cu[0],
              *matdim_cu = &memory_block_cu[2];

      auto kGridDim = new_dims[0];
      auto kBlockDim = std::min(int64_t(new_dims[1]), kMaxBlockDim);
      fill_grad_kernel<T><<<kGridDim, kBlockDim, 0, stream>>>(
          size, data, strides_cu, matdim_cu, offset, new_dims[0], new_dims[1]);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    fill_diagonal_tensor, ops::FillDiagonalTensorCUDAKernel<float>,
    ops::FillDiagonalTensorCUDAKernel<double>,
    ops::FillDiagonalTensorCUDAKernel<plat::float16>,
    ops::FillDiagonalTensorCUDAKernel<int>,
    ops::FillDiagonalTensorCUDAKernel<int64_t>,
    ops::FillDiagonalTensorCUDAKernel<int8_t>,
    ops::FillDiagonalTensorCUDAKernel<uint8_t>,
    ops::FillDiagonalTensorCUDAKernel<paddle::platform::complex<float>>,
    ops::FillDiagonalTensorCUDAKernel<paddle::platform::complex<double>>,
    ops::FillDiagonalTensorCUDAKernel<bool>);

REGISTER_OP_CUDA_KERNEL(
    fill_diagonal_tensor_grad, ops::FillDiagonalTensorGradCUDAKernel<float>,
    ops::FillDiagonalTensorGradCUDAKernel<double>,
    ops::FillDiagonalTensorGradCUDAKernel<int>,
    ops::FillDiagonalTensorGradCUDAKernel<int64_t>,
    ops::FillDiagonalTensorGradCUDAKernel<plat::float16>,
    ops::FillDiagonalTensorGradCUDAKernel<int8_t>,
    ops::FillDiagonalTensorGradCUDAKernel<uint8_t>,
    ops::FillDiagonalTensorGradCUDAKernel<paddle::platform::complex<float>>,
    ops::FillDiagonalTensorGradCUDAKernel<paddle::platform::complex<double>>,
    ops::FillDiagonalTensorGradCUDAKernel<bool>);
