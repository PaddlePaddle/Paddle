#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/roi_pool_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T>
__global__ void GPUROIPoolForward(
    const int nthreads, const T* input_data, const T* input_rois,
    const float spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    int* roi_batch_id_data, T* output_data, int64_t* argmax_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % channels;
    int n = i / pooled_width / pooled_height / channels;

    const T* offset_input_rois = input_rois + n * kROISize;
    int roi_batch_ind = roi_batch_id_data[n];
    int roi_start_w = round(offset_input_rois[0] * spatial_scale);
    int roi_start_h = round(offset_input_rois[1] * spatial_scale);
    int roi_end_w = round(offset_input_rois[2] * spatial_scale);
    int roi_end_h = round(offset_input_rois[3] * spatial_scale);

    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    int hstart = static_cast<int>(floor(static_cast<double>(ph) *
                                        static_cast<double>(roi_height) /
                                        static_cast<double>(pooled_height)));
    int wstart = static_cast<int>(floor(static_cast<double>(pw) *
                                        static_cast<double>(roi_width) /
                                        static_cast<double>(pooled_width)));
    int hend = static_cast<int>(ceil(static_cast<double>(ph + 1) *
                                     static_cast<double>(roi_height) /
                                     static_cast<double>(pooled_height)));
    int wend = static_cast<int>(ceil(static_cast<double>(pw + 1) *
                                     static_cast<double>(roi_width) /
                                     static_cast<double>(pooled_width)));
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    T maxval = is_empty ? 0 : -std::numeric_limits<T>::max();
    int maxidx = -1;
    const T* offset_input_data =
        input_data + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_data_index = h * width + w;
        if (offset_input_data[input_data_index] > maxval) {
          maxval = offset_input_data[input_data_index];
          maxidx = input_data_index;
        }
      }
    }
    output_data[i] = maxval;
    if (argmax_data) {
      argmax_data[i] = maxidx;
    }
  }
}

template <typename T>
__global__ void GPUROIPoolBackward(
    const int nthreads, const T* input_rois, const T* output_grad,
    const int64_t* argmax_data, const int num_rois, const float spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, int* roi_batch_id_data,
    T* input_grad) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % channels;
    int n = i / pooled_width / pooled_height / channels;

    int roi_batch_ind = roi_batch_id_data[n];
    int input_offset = (roi_batch_ind * channels + c) * height * width;
    int output_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_output_grad = output_grad + output_offset;
    T* offset_input_grad = input_grad + input_offset;
    const int64_t* offset_argmax_data = argmax_data + output_offset;

    int argmax = offset_argmax_data[ph * pooled_width + pw];
    if (argmax != -1) {
      platform::CudaAtomicAdd(
          offset_input_grad + argmax,
          static_cast<T>(offset_output_grad[ph * pooled_width + pw]));
    }
  }
}

template <typename Place, typename T>
class GPUROIPoolOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");
    auto* out = ctx.Output<Tensor>("Out");
    auto* argmax = ctx.Output<Tensor>("Argmax");

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    auto in_dims = in->dims();
    int batch_size = in_dims[0];
    auto in_stride = framework::stride(in_dims);
    int channels = in_dims[1];
    int height = in_dims[2];
    int width = in_dims[3];

    int rois_num = rois->dims()[0];

    if (rois_num == 0) return;

    int output_size = out->numel();
    int blocks = NumBlocks(output_size);
    int threads = kNumCUDAThreads;

    framework::Tensor roi_batch_id_list;
    roi_batch_id_list.Resize({rois_num});
    auto cplace = platform::CPUPlace();
    int* roi_batch_id_data = roi_batch_id_list.mutable_data<int>(cplace);
    auto rois_lod = rois->lod().back();
    int rois_batch_size = rois_lod.size() - 1;
    PADDLE_ENFORCE_EQ(
        rois_batch_size, batch_size,
        "The rois_batch_size and imgs batch_size must be the same.");
    int rois_num_with_lod = rois_lod[rois_batch_size];
    PADDLE_ENFORCE_EQ(rois_num, rois_num_with_lod,
                      "The rois_num from input and lod must be the same.");
    for (int n = 0; n < rois_batch_size; ++n) {
      for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
        roi_batch_id_data[i] = n;
      }
    }

    auto& dev_ctx = ctx.cuda_device_context();
    int bytes = roi_batch_id_list.numel() * sizeof(int);
    auto roi_ptr = memory::Alloc(dev_ctx, bytes);
    int* roi_id_data = reinterpret_cast<int*>(roi_ptr->ptr());
    const auto gplace = boost::get<platform::CUDAPlace>(ctx.GetPlace());
    memory::Copy(gplace, roi_id_data, cplace, roi_batch_id_data, bytes,
                 dev_ctx.stream());

    GPUROIPoolForward<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
        output_size, in->data<T>(), rois->data<T>(), spatial_scale, channels,
        height, width, pooled_height, pooled_width, roi_id_data,
        out->mutable_data<T>(ctx.GetPlace()),
        argmax->mutable_data<int64_t>(ctx.GetPlace()));
  }
};

template <typename Place, typename T>
class GPUROIPoolGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");
    auto* argmax = ctx.Input<Tensor>("Argmax");

    auto* out_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* x_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    int rois_num = rois->dims()[0];
    int channels = in->dims()[1];
    int height = in->dims()[2];
    int width = in->dims()[3];

    if (x_grad) {
      framework::Tensor roi_batch_id_list;
      roi_batch_id_list.Resize({rois_num});
      auto cplace = platform::CPUPlace();
      int* roi_batch_id_data = roi_batch_id_list.mutable_data<int>(cplace);
      auto rois_lod = rois->lod().back();
      int rois_batch_size = rois_lod.size() - 1;
      for (int n = 0; n < rois_batch_size; ++n) {
        for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
          roi_batch_id_data[i] = n;
        }
      }

      auto& dev_ctx = ctx.cuda_device_context();
      int bytes = roi_batch_id_list.numel() * sizeof(int);
      auto roi_ptr = memory::Alloc(dev_ctx, bytes);
      int* roi_id_data = reinterpret_cast<int*>(roi_ptr->ptr());
      const auto gplace = boost::get<platform::CUDAPlace>(ctx.GetPlace());
      memory::Copy(gplace, roi_id_data, cplace, roi_batch_id_data, bytes,
                   dev_ctx.stream());

      x_grad->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<Place, T> set_zero;
      set_zero(dev_ctx, x_grad, static_cast<T>(0));

      int output_grad_size = out_grad->numel();
      int blocks = NumBlocks(output_grad_size);
      int threads = kNumCUDAThreads;

      if (output_grad_size > 0) {
        GPUROIPoolBackward<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            output_grad_size, rois->data<T>(), out_grad->data<T>(),
            argmax->data<int64_t>(), rois_num, spatial_scale, channels, height,
            width, pooled_height, pooled_width, roi_id_data,
            x_grad->mutable_data<T>(ctx.GetPlace()));
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roi_pool,
    ops::GPUROIPoolOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUROIPoolOpKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    roi_pool_grad,
    ops::GPUROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, double>);
