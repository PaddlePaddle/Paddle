#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/roi_pool_op.h"
#include "paddle/fluid/platform/cuda_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;
static constexpr int kROISize = 5;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T>
__global__ void GPUROIPoolForward(const int nthreads, const T* input_data,
                                  const int64_t* input_rois,
                                  const float spatial_scale, const int channels,
                                  const int height, const int width,
                                  const int pooled_height,
                                  const int pooled_width, T* output_data,
                                  int64_t* argmax_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const int64_t* offset_input_rois = input_rois + n * kROISize;
    int roi_batch_ind = offset_input_rois[0];
    int roi_start_w = round(offset_input_rois[1] * spatial_scale);
    int roi_start_h = round(offset_input_rois[2] * spatial_scale);
    int roi_end_w = round(offset_input_rois[3] * spatial_scale);
    int roi_end_h = round(offset_input_rois[4] * spatial_scale);

    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<T>(ph) * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw) * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1) * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1) * bin_size_w));

    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    T maxval = is_empty ? 0 : -std::numeric_limits<T>::max();
    int maxidx = -1;
    const T* offset_input_data =
        input_data + (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_data_index = h * width + w;
        if (offset_input_data[input_data_index] > maxval) {
          maxval = offset_input_data[input_data_index];
          maxidx = input_data_index;
        }
      }
    }
    output_data[index] = maxval;
    if (argmax_data) {
      argmax_data[index] = maxidx;
    }
  }
}

template <typename T>
__global__ void GPUROIPoolBackward(
    const int nthreads, const int64_t* input_rois, const T* output_grad,
    const int64_t* argmax_data, const int num_rois, const float spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, T* input_grad) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const int64_t* offset_input_rois = input_rois + n * kROISize;
    int roi_batch_ind = offset_input_rois[0];
    int input_offset = (roi_batch_ind * channels + c) * height * width;
    int output_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_output_grad = output_grad + output_offset;
    T* offset_input_grad = input_grad + input_offset;
    const int64_t* offset_argmax_data = argmax_data + output_offset;

    int argmax = offset_argmax_data[ph * pooled_width + pw];
    if (argmax != -1) {
      platform::CudaAtomicAdd(
          offset_input_grad + argmax,
          static_cast<T>(offset_output_grad[ph * pooled_width + pw]));
    }
  }
}

template <typename Place, typename T>
class GPUROIPoolOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<Tensor>("ROIs");
    auto* out = ctx.Output<Tensor>("Out");
    auto* argmax = ctx.Output<Tensor>("Argmax");

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    auto in_dims = in->dims();
    auto in_stride = framework::stride(in_dims);
    int channels = in_dims[1];
    int height = in_dims[2];
    int width = in_dims[3];

    size_t rois_num = rois->dims()[0];
    if (rois_num == 0) return;

    int output_size = out->numel();
    int blocks = NumBlocks(output_size);
    int threads = kNumCUDAThreads;

    GPUROIPoolForward<
        T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        output_size, in->data<T>(), rois->data<int64_t>(), spatial_scale,
        channels, height, width, pooled_height, pooled_width,
        out->mutable_data<T>(ctx.GetPlace()),
        argmax->mutable_data<int64_t>(ctx.GetPlace()));
  }
};

template <typename Place, typename T>
class GPUROIPoolGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<Tensor>("ROIs");
    auto* argmax = ctx.Input<Tensor>("Argmax");

    auto* out_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* x_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    size_t rois_num = rois->dims()[0];
    int channels = in->dims()[1];
    int height = in->dims()[2];
    int width = in->dims()[3];

    if (x_grad) {
      x_grad->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<Place, T> set_zero;
      set_zero(ctx.cuda_device_context(), x_grad, static_cast<T>(0));

      int output_grad_size = out_grad->numel();
      int blocks = NumBlocks(output_grad_size);
      int threads = kNumCUDAThreads;

      if (output_grad_size > 0) {
        GPUROIPoolBackward<
            T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
            output_grad_size, rois->data<int64_t>(), out_grad->data<T>(),
            argmax->data<int64_t>(), rois_num, spatial_scale, channels, height,
            width, pooled_height, pooled_width,
            x_grad->mutable_data<T>(ctx.GetPlace()));
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roi_pool,
    ops::GPUROIPoolOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUROIPoolOpKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    roi_pool_grad,
    ops::GPUROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUROIPoolOpKernel<paddle::platform::CUDADeviceContext, double>);
