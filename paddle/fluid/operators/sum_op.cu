#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/platform/device_context.h>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/sum_op.h"
#include "paddle/fluid/platform/float16.h"

namespace plat = paddle::platform;

namespace paddle {
namespace operators {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <class T>
__global__ void sum_gpu(const T *in_0, const T *in_1, T *out, int64_t N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    out[id] = in_0[id] + in_1[id];
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void sum_gpu_array(T **in, T *out, int64_t N, size_t in_size,
                              bool read_dst) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    T total(0);
    for (int i = 0; i < in_size; ++i) {
      const T *tmp = in[i];
      if (tmp != nullptr) {
        total += tmp[id];
      }
    }
    if (read_dst) {
      out[id] += total;
    } else {
      out[id] = total;
    }
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void sum_gpu_sr(T **sr_in, T **sr_out, int64_t N, size_t rows) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    for (int i = 0; i < rows; ++i) {
      const T *tmp = sr_in[i];
      T *tmp_out = sr_out[i];
      if (tmp != nullptr && tmp_out != nullptr) {
        tmp_out[id] += tmp[id];
      }
    }
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void sum_gpu4(const T *in_0, const T *in_1, T *out, int64_t N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = id; i < N / 4; i += blockDim.x * gridDim.x) {
    float4 *in0_4 = reinterpret_cast<float4 *>(const_cast<T *>(in_0));
    float4 *in1_4 = reinterpret_cast<float4 *>(const_cast<T *>(in_1));
    float4 tmp;
    tmp.x = in0_4[i].x + in1_4[i].x;
    tmp.y = in0_4[i].y + in1_4[i].y;
    tmp.z = in0_4[i].z + in1_4[i].z;
    tmp.w = in0_4[i].w + in1_4[i].w;
    reinterpret_cast<float4 *>(out)[i] = tmp;
  }
}

template <class T>
void FuseSumCompute(const framework::ExecutionContext &context) {
  auto in_vars = context.MultiInputVar("X");
  const size_t in_num = in_vars.size();

  constexpr size_t theory_sm_threads = 1024;
  auto &dev_ctx =
      context.template device_context<platform::CUDADeviceContext>();
  auto stream = dev_ctx.stream();

  auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  auto sm_count = max_threads / theory_sm_threads;
  size_t tile_size = 0;
  dim3 grids;
  dim3 blocks;

  auto KeCompute = [&](size_t length) {
    if (length >= max_threads)
      tile_size = 1024;
    else if (length < max_threads && length > sm_count * 128)
      tile_size = 512;
    else if (length <= sm_count * 128)
      tile_size = 256;
    grids = dim3(CEIL_DIV(length, tile_size), 1, 1);
    blocks = dim3(tile_size, 1, 1);
  };

  auto *out = context.Output<LoDTensor>("Out");

  auto out_var = context.OutputVar("Out");
  bool in_place = in_vars[0] == out_var;

  if (!in_place) {
    out->mutable_data<T>(context.GetPlace());
  }
  int start = in_place ? 1 : 0;
  if (!in_place) {
    if (in_num == 2 && in_vars[0]->IsType<framework::LoDTensor>() &&
        in_vars[1]->IsType<framework::LoDTensor>()) {
      auto &in_0 = in_vars[0]->Get<framework::LoDTensor>();
      auto &in_1 = in_vars[1]->Get<framework::LoDTensor>();

      auto length = in_0.numel();
      if (length) {
        KeCompute(length);
        sum_gpu<T><<<grids, blocks, 0, stream>>>(in_0.data<T>(), in_1.data<T>(),
                                                 out->data<T>(), length);
      } else {
        math::SetConstant<platform::CUDADeviceContext, T> constant_functor;
        constant_functor(
            context.template device_context<platform::CUDADeviceContext>(), out,
            static_cast<T>(0));
      }
      return;
    }
  }

  std::vector<const T *> in_data;
  std::vector<int> selectrow_index;
  int64_t lod_length = 0;
  bool dst_write = false;
  for (int i = start; i < in_num; ++i) {
    if (in_vars[i]->IsType<framework::LoDTensor>()) {
      auto &in_i = in_vars[i]->Get<framework::LoDTensor>();
      in_data.emplace_back(in_i.data<T>());
      lod_length = in_i.numel();
    } else if (in_vars[i]->IsType<framework::SelectedRows>()) {
      selectrow_index.push_back(i);
    }
  }

  // compute select rows seperately.
  if (!selectrow_index.empty()) {
    std::vector<const T *> out_data;
    std::vector<const T *> sr_in_data;
    size_t rows = 0;
    int64_t length = 0;
    for (auto index : selectrow_index) {
      auto &sr = in_vars[index]->Get<framework::SelectedRows>();
      auto &sr_value = sr.value();
      auto &sr_rows = sr.rows();

      auto row_numel = sr_value.numel() / sr_rows.size();
      auto out_dims = out->dims();

      PADDLE_ENFORCE_EQ(sr.height(), out_dims[0]);
      PADDLE_ENFORCE_EQ(row_numel, out->numel() / sr.height());

      auto *sr_data = sr_value.data<T>();
      auto *sr_out_data = out->data<T>();
      rows += sr_rows.size();
      length = row_numel;

      for (size_t i = 0; i < sr_rows.size(); ++i) {
        sr_in_data.emplace_back(&sr_data[i * row_numel]);
        out_data.emplace_back(&sr_out_data[sr_rows[i] * row_numel]);
      }
    }
    if (!sr_in_data.empty() && !out_data.empty()) {
      auto tmp_sr_in_array =
          platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx).Allocate(
              sr_in_data.size() * sizeof(T *));

      memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                   tmp_sr_in_array->ptr(), platform::CPUPlace(),
                   reinterpret_cast<void *>(sr_in_data.data()),
                   sr_in_data.size() * sizeof(T *), dev_ctx.stream());

      T **sr_in_array_data = reinterpret_cast<T **>(tmp_sr_in_array->ptr());

      auto tmp_out_array =
          platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx).Allocate(
              out_data.size() * sizeof(T *));

      memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                   tmp_out_array->ptr(), platform::CPUPlace(),
                   reinterpret_cast<void *>(out_data.data()),
                   out_data.size() * sizeof(T *), dev_ctx.stream());

      T **out_array_data = reinterpret_cast<T **>(tmp_out_array->ptr());
      KeCompute(length);
      sum_gpu_sr<T><<<grids, blocks, 0, stream>>>(sr_in_array_data,
                                                  out_array_data, length, rows);
      dst_write = true;
    }
  }
  // if indata not null, merge into one kernel call.
  if (!in_data.empty()) {
    auto tmp_in_array =
        platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx).Allocate(
            in_data.size() * sizeof(T *));

    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_in_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(in_data.data()),
                 in_data.size() * sizeof(T *), dev_ctx.stream());

    T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());
    KeCompute(lod_length);
    sum_gpu_array<T><<<grids, blocks, 0, stream>>>(
        in_array_data, out->data<T>(), lod_length, in_data.size(), dst_write);
  }
}

template <typename T>
class SumKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto in_vars = context.MultiInputVar("X");
    const size_t in_num = in_vars.size();
    auto out_var = context.OutputVar("Out");

    bool in_place = out_var == in_vars[0];
    if (out_var->IsType<framework::LoDTensor>()) {
      FuseSumCompute<T>(context);
    } else if (out_var->IsType<framework::SelectedRows>()) {
      if (in_place && in_vars.size() < 2) {
        return;
      }

      std::vector<const paddle::framework::SelectedRows *> inputs;
      SelectedRows temp_in0;

      if (in_place) {
        auto &in0 = in_vars[0]->Get<SelectedRows>();
        temp_in0.set_height(in0.height());
        temp_in0.set_rows(in0.rows());
        framework::TensorCopy(in0.value(), in0.place(),
                              context.device_context(),
                              temp_in0.mutable_value());
        inputs.push_back(&temp_in0);
        for (size_t i = 1; i < in_vars.size(); ++i) {
          auto &in = in_vars[i]->Get<SelectedRows>();
          if (in.rows().size() > 0) {
            inputs.push_back(&in);
          }
        }
      } else {
        for (auto &in_var : in_vars) {
          auto &in = in_var->Get<SelectedRows>();
          if (in.rows().size() > 0) {
            inputs.push_back(&in_var->Get<SelectedRows>());
          }
        }
      }

      auto *out = context.Output<SelectedRows>("Out");
      out->mutable_rows()->clear();

      bool has_data = false;
      for (auto &in : inputs) {
        if (in->rows().size() > 0) {
          has_data = true;
          break;
        }
      }
      if (has_data) {
        math::scatter::MergeAdd<platform::CUDADeviceContext, T> merge_add;
        merge_add(
            context.template device_context<platform::CUDADeviceContext>(),
            inputs, out);

        out->SyncIndex();

      } else {
        // no data, just set a empty out tensor.
        out->mutable_value()->mutable_data<T>(framework::make_ddim({0}),
                                              context.GetPlace());
      }
    } else if (out_var->IsType<framework::LoDTensorArray>()) {
      auto &out_array = *out_var->GetMutable<framework::LoDTensorArray>();
      for (size_t i = in_place ? 1 : 0; i < in_vars.size(); ++i) {
        PADDLE_ENFORCE(in_vars[i]->IsType<framework::LoDTensorArray>(),
                       "Only support all inputs are TensorArray");
        auto &in_array = in_vars[i]->Get<framework::LoDTensorArray>();

        for (size_t i = 0; i < in_array.size(); ++i) {
          if (in_array[i].numel() != 0) {
            if (i >= out_array.size()) {
              out_array.resize(i + 1);
            }
            if (out_array[i].numel() == 0) {
              framework::TensorCopy(in_array[i], in_array[i].place(),
                                    context.device_context(), &out_array[i]);
              out_array[i].set_lod(in_array[i].lod());
            } else {
              PADDLE_ENFORCE(out_array[i].lod() == in_array[i].lod());
              auto in = EigenVector<T>::Flatten(in_array[i]);
              auto result = EigenVector<T>::Flatten(out_array[i]);
              result.device(
                  *context
                       .template device_context<platform::CUDADeviceContext>()
                       .eigen_device()) = result + in;
            }
          }
        }
      }
    } else {
      PADDLE_THROW("Unexpected branch, output variable type is %s",
                   framework::ToTypeName(out_var->Type()));
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    sum, ops::SumKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SumKernel<paddle::platform::CUDADeviceContext, double>,
    ops::SumKernel<paddle::platform::CUDADeviceContext, int>,
    ops::SumKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::SumKernel<paddle::platform::CUDADeviceContext, plat::float16>);
