// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/uniform_random_op.h"

namespace paddle {
namespace operators {

template <typename T>
class GPURandintKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    std::vector<int64_t> new_shape;
    auto list_new_shape_tensor =
        context.MultiInput<framework::Tensor>("ShapeTensorList");
    if (list_new_shape_tensor.size() > 0 || context.HasInput("ShapeTensor")) {
      if (context.HasInput("ShapeTensor")) {
        auto* shape_tensor = context.Input<framework::Tensor>("ShapeTensor");
        new_shape = GetNewDataFromShapeTensor(shape_tensor);
      } else if (list_new_shape_tensor.size() > 0) {
        new_shape = GetNewDataFromShapeTensorList(list_new_shape_tensor);
      }
    }

    platform::CPUPlace cpu;
    auto dtype = static_cast<framework::proto::VarType::Type>(
        context.Attr<int>("dtype"));
    auto* out = context.Output<framework::LoDTensor>("Out");
    if (!new_shape.empty()) out->Resize(framework::make_ddim(new_shape));
    T low = static_cast<T>(context.Attr<int>("low"));
    T high = static_cast<T>(context.Attr<int>("high")) - 1;
    framework::LoDTensor tensor;
    tensor.Resize(out->dims());
    tensor.mutable_data(cpu, dtype);
    T* data = tensor.mutable_data<T>(cpu);

    int64_t size = out->numel();
    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    std::minstd_rand engine;
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
    }
    engine.seed(seed);
    std::uniform_int_distribution<> dist(context.Attr<int>("low"),
                                         context.Attr<int>("high") - 1);
    for (int64_t i = 0; i < size; ++i) data[i] = dist(engine);

    if (platform::is_gpu_place(context.GetPlace())) {
      // Copy tensor to out
      framework::TensorCopy(tensor, context.GetPlace(), out);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(randint, ops::GPURandintKernel<int>,
                        ops::GPURandintKernel<int64_t>)
