#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <vector>
#include "paddle/fluid/operators/masked_select_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__global__ void MaskedSelect(const int nums, const T* input_data,
                             const bool* mask_data, T* output_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  int j = index;
  for (size_t i = index; i < nums; i += offset) {
    if (mask_data[i]) {
      output_data[j] = input_data[i];
      j += offset;
    }
  }
}

template <typename DeviceContext, typename T>
class MaskedSelectOPCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<framework::Tensor>("input");
    auto* mask = ctx.Input<framework::Tensor>("mask");
    auto* output = ctx.Output<framework::Tensor>("Out");

    const T* input_data = input->data<T>();
    auto* mask_data = mask->data<bool>();

    int out_dim = 0;
    for (size_t i = 0; i < mask->numel(); i++) {
      if (mask_data[i]) {
        out_dim++;
      }
    }
    output->Resize({out_dim});

    T* output_data = output->mutable_data<T>(ctx.GetPlace());

    int blocks = NumBlocks(input->numel());
    int threads = kNumCUDAThreads;

    MaskedSelect<T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        input->numel(), input_data, mask_data, output_data);
  }
};

template <typename T>
__global__ void MaskedSelectGrad(const int nums, const T* output_grad_data,
                                 const bool* mask_data, T* input_grad_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  int j = index;
  for (size_t i = index; i < nums; i += offset) {
    if (mask_data[i]) {
      input_grad_data[i] = output_grad_data[j];
      j += offset;
    } else {
      input_grad_data[i] = 0;
    }
  }
}

template <typename DeviceContext, typename T>
class MaskedSelectGradOPCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* output_grad =
        ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* input_grad =
        ctx.Output<framework::Tensor>(framework::GradVarName("input"));
    auto* mask = ctx.Input<framework::Tensor>("mask");

    // input_grad->Resize({mask->dims()})

    int blocks = NumBlocks(mask->numel());
    int threads = kNumCUDAThreads;

    auto* mask_data = mask->data<bool>();
    const T* output_grad_data = output_grad->data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());

    MaskedSelectGrad<
        T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        blocks, output_grad_data, mask_data, input_grad_data);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    masked_select,
    ops::MaskedSelectOPCUDAKernel<plat::CUDADeviceContext, float>,
    ops::MaskedSelectOPCUDAKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    masked_select_grad,
    ops::MaskedSelectGradOPCUDAKernel<plat::CUDADeviceContext, float>,
    ops::MaskedSelectGradOPCUDAKernel<plat::CUDADeviceContext, double>);
