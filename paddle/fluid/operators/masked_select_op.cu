#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/scan.h>
#include "paddle/fluid/operators/masked_select_op.h"
namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;
using DDim = framework::DDim;

__global__ void SetMaskArray(const bool* mask, int32_t* mask_array, int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx])
      mask_array[idx] = 1;
    else
      mask_array[idx] = 0;
  }
}

template <typename T>
__global__ void SelectWithPrefixMask(const int32_t* mask_prefix_sum,
                                     const bool* mask, const T* input, T* out,
                                     int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx]) {
      int index = mask_prefix_sum[idx];
      out[index] = input[idx];
    }
  }
}

template <typename T>
__global__ void SelectGradWithPrefixMask(const int32_t* mask_prefix_sum,
                                         const bool* mask, const T* input,
                                         T* out, int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx]) {
      int index = mask_prefix_sum[idx];
      out[idx] = input[index];
    } else {
      out[idx] = 0;
    }
  }
}

template <typename DeviceContext, typename T>
class MaskedSelectCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto input = ctx.Input<framework::Tensor>("X");
    auto mask = ctx.Input<framework::Tensor>("Mask");
    auto out = ctx.Output<framework::Tensor>("Y");
    auto* mask_data = mask->data<bool>();
    auto input_data = input->data<T>();

    auto mask_size = mask->numel();
    auto input_dim = input->dims();
    auto mask_dim = mask->dims();
    PADDLE_ENFORCE_EQ(
        input_dim, mask_dim,
        platform::errors::InvalidArgument(
            "The dim size of input and mask in OP(masked_selected) "
            "must be equal, but got input dim:(%ld), mask dim: "
            "(%ld). Please check input "
            "value.",
            input_dim, mask_dim));

    thrust::device_ptr<const bool> mask_dev_ptr =
        thrust::device_pointer_cast(mask_data);
    thrust::device_vector<T> mask_vec(mask_dev_ptr, mask_dev_ptr + mask_size);
    auto out_size = thrust::count(mask_vec.begin(), mask_vec.end(), true);

    framework::DDim out_dim{out_size};
    out->Resize(out_dim);
    auto out_data = out->mutable_data<T>(ctx.GetPlace());

    Tensor mask_array;
    Tensor mask_prefix_sum;
    mask_array.Resize(mask_dim);
    mask_prefix_sum.Resize(mask_dim);

    int32_t* mask_array_data = mask_array.mutable_data<int32_t>(ctx.GetPlace());
    int32_t* mask_prefix_sum_data =
        mask_prefix_sum.mutable_data<int32_t>(ctx.GetPlace());
    int threads = 512;
    int grid = (mask_size + threads - 1) / threads;
    auto stream = ctx.cuda_device_context().stream();
    SetMaskArray<<<grid, threads, 0, stream>>>(mask_data, mask_array_data,
                                               mask_size);

    thrust::device_ptr<int32_t> mask_array_dev_ptr =
        thrust::device_pointer_cast(mask_array_data);
    thrust::device_vector<int32_t> mask_array_vec(
        mask_array_dev_ptr, mask_array_dev_ptr + mask_size);
    thrust::exclusive_scan(thrust::device, mask_array_vec.begin(),
                           mask_array_vec.end(), mask_prefix_sum_data);

    SelectWithPrefixMask<T><<<grid, threads, 0, stream>>>(
        mask_prefix_sum_data, mask_data, input_data, out_data, mask_size);
  }
};

template <typename DeviceContext, typename T>
class MaskedSelectGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto input = ctx.Input<framework::Tensor>(framework::GradVarName("Y"));
    auto mask = ctx.Input<framework::Tensor>("Mask");
    auto out = ctx.Output<framework::Tensor>(framework::GradVarName("X"));
    auto* mask_data = mask->data<bool>();
    auto* input_data = input->data<T>();
    auto* out_data = out->mutable_data<T>(ctx.GetPlace());

    auto input_size = input->numel();
    auto mask_size = mask->numel();
    auto mask_dim = mask->dims();

    auto out_size = mask_size;

    Tensor mask_array;
    Tensor mask_prefix_sum;
    mask_array.Resize(mask_dim);
    mask_prefix_sum.Resize(mask_dim);

    int32_t* mask_array_data = mask_array.mutable_data<int32_t>(ctx.GetPlace());
    int32_t* mask_prefix_sum_data =
        mask_prefix_sum.mutable_data<int32_t>(ctx.GetPlace());
    int threads = 512;
    int grid = (mask_size + threads - 1) / threads;
    auto stream = ctx.cuda_device_context().stream();
    SetMaskArray<<<grid, threads, 0, stream>>>(mask_data, mask_array_data,
                                               mask_size);

    thrust::device_ptr<int32_t> mask_array_dev_ptr =
        thrust::device_pointer_cast(mask_array_data);
    thrust::device_vector<int32_t> mask_array_vec(
        mask_array_dev_ptr, mask_array_dev_ptr + mask_size);
    thrust::exclusive_scan(thrust::device, mask_array_vec.begin(),
                           mask_array_vec.end(), mask_prefix_sum_data);

    SelectGradWithPrefixMask<T><<<grid, threads, 0, stream>>>(
        mask_prefix_sum_data, mask_data, input_data, out_data, mask_size);
  }
};
}  // namespace operators
}  // namespace paddle
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    masked_select,
    ops::MaskedSelectCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MaskedSelectCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MaskedSelectCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::MaskedSelectCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    masked_select_grad,
    ops::MaskedSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MaskedSelectGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                    double>,
    ops::MaskedSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::MaskedSelectGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                    int64_t>);
