#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the spopecific language governing permissions and
limitations under the License. */

#include <utility>
#include <vector>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#include "paddle/fluid/operators/conv_cudnn_op_cache.h"
#include "paddle/fluid/operators/conv_op.h"
#include "paddle/fluid/operators/math/padding.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/cudnn_workspace_helper.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/profiler.h"

DECLARE_bool(cudnn_deterministic);
DECLARE_uint64(conv_workspace_size_limit);
DECLARE_bool(cudnn_exhaustive_search);

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using DataLayout = platform::DataLayout;
template <typename T>
using ScalingParamType = typename platform::CudnnDataType<T>::ScalingParamType;
using framework::AlgorithmsCache;

static inline void GetNCDHW(const framework::DDim& dims,
                            const DataLayout& layout, int* N, int* C, int* D,
                            int* H, int* W) {
  *N = dims[0];
  *C = layout == DataLayout::kNCHW ? dims[1] : dims[dims.size() - 1];
  int i = layout == DataLayout::kNCHW ? 0 : 1;
  if (dims.size() == 5) {
    *D = dims[2 - i];
    *H = dims[3 - i];
    *W = dims[4 - i];
  } else {
    *D = 1;
    *H = dims[2 - i];
    *W = dims[3 - i];
  }
}

template <typename DeviceContext, typename T, size_t D>
static void Slice_2(const framework::ExecutionContext& context,
                    const Tensor* input, Tensor* out,
                    const std::vector<int>& starts,
                    const std::vector<int>& axes) {
  auto& place =
      *context.template device_context<DeviceContext>().eigen_device();
  auto in_dims = input->dims();
  auto new_out_dims = out->dims();
  auto offsets = Eigen::array<int, D>();
  auto extents = Eigen::array<int, D>();
  for (size_t i = 0; i < D; ++i) {
    offsets[i] = 0;
    extents[i] = new_out_dims[i];
  }

  int start;
  for (size_t i = 0; i < axes.size(); ++i) {
    start = starts[i];
    if (start < 0) {
      start = (start + in_dims[axes[i]]);
    }
    start = std::max(start, 0);
    offsets[axes[i]] = start;
  }
  auto in_t =
      framework::EigenTensor<T, D, Eigen::RowMajor, Eigen::DenseIndex>::From(
          *input);

  auto out_t =
      framework::EigenTensor<T, D, Eigen::RowMajor, Eigen::DenseIndex>::From(
          *out, new_out_dims);
  out_t.device(place) = in_t.slice(offsets, extents);
}

template <typename T>
class CUDNNConvOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "It must use CUDAPlace.");
    const Tensor* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* output = ctx.Output<Tensor>("Output");
    output->mutable_data<T>(ctx.GetPlace());
    const std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");

    if (exhaustive_search && FLAGS_cudnn_deterministic) {
      PADDLE_THROW(
          "Cann't set exhaustive_search True and "
          "FLAGS_cudnn_deterministic True at same time.");
    }
    const std::string padding_algorithm =
        ctx.Attr<std::string>("padding_algorithm");
    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    // ------------ transformed tensor -----------
    Tensor transformed_input_channel(input->type());
    Tensor transformed_output(output->type());
    T* output_data = nullptr;
    if (channel_last) {
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, output,
                                                           &transformed_output);

    } else {
      transformed_input_channel = *input;
      transformed_output = *output;
    }
    output_data = transformed_output.data<T>();

    // update padding and dilation
    auto in_dims = transformed_input_channel.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());

    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    Tensor transformed_input;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_input_channel.dims()[0];
      new_input_shape_vec[1] = transformed_input_channel.dims()[1];

      std::vector<int> input_pad(transformed_input_channel.dims().size() * 2,
                                 0);
      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            transformed_input_channel.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      const int rank = transformed_input_channel.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        default:
          PADDLE_THROW("ConvOp only support tensors with 4 or 5 dimensions.");
      }

    } else {
      transformed_input = transformed_input_channel;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = transformed_input.data<T>();
    const T* filter_data = filter->data<T>();

    // ------------------- cudnn descriptors ---------------------
    ConvArgs args{&transformed_input, filter,   &transformed_output, strides,
                  padding_common,     dilations};

    auto handle = dev_ctx.cudnn_handle();
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    auto dtype = platform::CudnnDataType<T>::type;
    DataLayout layout = DataLayout::kNCHW;
    if (transformed_input_channel.dims().size() == 5) {
      layout = DataLayout::kNCDHW;
    }
    auto layout_format = GetCudnnTensorFormat(layout);

    args.handle = handle;
    args.cdesc.set(dtype, padding_common, strides, dilations);

#if CUDNN_VERSION_MIN(7, 0, 1)
    // cudnn 7 can support groups, no need to do it manually
    // FIXME(typhoonzero): find a better way to disable groups
    // rather than setting it to 1.
    CUDNN_ENFORCE(platform::dynload::hipdnnSetConvolutionGroupCount(
        args.cdesc.desc(), groups));
    groups = 1;
#endif
    args.idesc.set(transformed_input, groups);

    args.wdesc.set(*filter, layout_format, groups);
    args.odesc.set(transformed_output, groups);
    int i_n, i_c, i_d, i_h, i_w;

    GetNCDHW(transformed_input.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d,
             &i_h, &i_w);
    int o_n, o_c, o_d, o_h, o_w;
    GetNCDHW(transformed_output.dims(), DataLayout::kNCHW, &o_n, &o_c, &o_d,
             &o_h, &o_w);

    int group_offset_in = i_c / groups * i_h * i_w * i_d;
    int group_offset_out = o_c / groups * o_h * o_w * o_d;
    int group_offset_filter = filter->numel() / groups;
    // ------------------- cudnn conv workspace ---------------------
    size_t workspace_size = 0;  // final workspace to allocate.
    // ------------------- cudnn conv algorithm ---------------------
    hipdnnConvolutionFwdAlgo_t algo{};

    using search = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
    algo = search::Find<T>(args, exhaustive_search, false, 0, ctx);
    workspace_size = search::GetWorkspaceSize(args, algo);

    // ------------------- cudnn conv forward ---------------------
    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    for (int i = 0; i < groups; i++) {
      workspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionForward(
                handle, &alpha, args.idesc.desc(),
                input_data + i * group_offset_in, args.wdesc.desc(),
                filter_data + i * group_offset_filter, args.cdesc.desc(), algo,
                workspace_ptr, workspace_size, &beta, args.odesc.desc(),
                output_data + i * group_offset_out));
          },
          workspace_size);
    }

    if (channel_last) {
      TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
          ctx, &transformed_output, output);
    }
  }
};

template <typename T>
class CUDNNConvGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "It must use CUDAPlace.");
    auto input = ctx.Input<Tensor>("Input");
    auto filter = ctx.Input<Tensor>("Filter");
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));

    const T* filter_data = filter->data<T>();
    if (input_grad) {
      input_grad->mutable_data<T>(ctx.GetPlace());
    }
    if (filter_grad) {
      filter_grad->mutable_data<T>(ctx.GetPlace());
    }

    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    int groups = ctx.Attr<int>("groups");
    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");
    bool deterministic = FLAGS_cudnn_deterministic;
    if (exhaustive_search && deterministic) {
      PADDLE_THROW(
          "Can't set exhaustive_search True and "
          "FLAGS_cudnn_deterministic True at same time.");
    }
    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    // transform Tensor
    Tensor transformed_input_channel(input->type());
    Tensor transformed_output_grad_channel(output_grad->type());
    Tensor transformed_input_grad_channel(input->type());

    if (channel_last) {
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, output_grad, &transformed_output_grad_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, output_grad, &transformed_output_grad_channel);

      if (input_grad) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, input_grad, &transformed_input_grad_channel);
      }

    } else {
      transformed_input_channel = *input;
      transformed_output_grad_channel = *output_grad;
      if (input_grad) {
        transformed_input_grad_channel.ShareDataWith(*input_grad);
      }
    }

    //  update paddings
    auto in_dims = transformed_input_channel.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    // cuDNN only supports padding the same amount on every dimension.
    // So we create a new padded input tensor.
    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);
    Tensor transformed_input(input->type());
    Tensor transformed_input_grad(input->type());
    std::vector<int> padding_common(data_dim, 0);
    std::vector<int> input_pad(transformed_input_channel.dims().size() * 2, 0);

    if (!is_sys_pad) {
      // get pad
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_input_channel.dims()[0];
      new_input_shape_vec[1] = transformed_input_channel.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            transformed_input_channel.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);

      transformed_input_grad.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      if (input_grad) {
        transformed_input_grad =
            ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
                new_input_shape, dev_ctx);
      }
      // pad for input
      const int rank = transformed_input_channel.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        default:
          PADDLE_THROW("ConvOp only support tensors with 4 or 5 dimensions.");
      }
    } else {
      transformed_input.ShareDataWith(transformed_input_channel);
      if (input_grad) {
        transformed_input_grad.ShareDataWith(transformed_input_grad_channel);
      }
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = transformed_input.data<T>();
    const T* output_grad_data = transformed_output_grad_channel.data<T>();
    T* filter_grad_data = nullptr;
    T* input_grad_data = nullptr;
    T* transformed_input_grad_data = nullptr;

    ConvArgs args1{&transformed_input_grad,
                   filter,
                   &transformed_output_grad_channel,
                   strides,
                   padding_common,
                   dilations};
    ConvArgs args2{&transformed_input,
                   filter_grad,
                   &transformed_output_grad_channel,
                   strides,
                   padding_common,
                   dilations};

    auto handle = dev_ctx.cudnn_handle();
    auto dtype = platform::CudnnDataType<T>::type;
    DataLayout layout = DataLayout::kNCHW;
    if (input->dims().size() == 5) {
      layout = DataLayout::kNCDHW;
    }
    auto layout_tensor = GetCudnnTensorFormat(layout);
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();

    int i_n, i_c, i_d, i_h, i_w;
    GetNCDHW(transformed_input.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d,
             &i_h, &i_w);
    int o_n, o_c, o_d, o_h, o_w;
    GetNCDHW(transformed_output_grad_channel.dims(), DataLayout::kNCHW, &o_n,
             &o_c, &o_d, &o_h, &o_w);

    int group_offset_in = i_c / groups * i_h * i_w * i_d;
    int group_offset_out = o_c / groups * o_h * o_w * o_d;
    int group_offset_filter = filter->numel() / groups;
    // ------------------- cudnn backward algorithm ---------------------
    hipdnnConvolutionBwdDataAlgo_t data_algo =
        static_cast<hipdnnConvolutionBwdDataAlgo_t>(0);
    hipdnnConvolutionBwdFilterAlgo_t filter_algo =
        static_cast<hipdnnConvolutionBwdFilterAlgo_t>(0);
    size_t workspace_size = 0;
    int iwo_groups, c_groups;

#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_groups = 1;
    c_groups = groups;
    groups = 1;
#endif

    if (input_grad) {
      // ------------------- cudnn descriptors ---------------------
      input_grad_data = input_grad->data<T>();
      transformed_input_grad_data = transformed_input_grad.data<T>();
      args1.handle = handle;
      args1.idesc.set(transformed_input_grad, iwo_groups);
      args1.wdesc.set(*filter, layout_tensor, iwo_groups);
      args1.odesc.set(transformed_output_grad_channel, iwo_groups);
      args1.cdesc.set(dtype, padding_common, strides, dilations, c_groups);

      using search1 = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
      data_algo =
          search1::Find<T>(args1, exhaustive_search, deterministic, 0, ctx);
      workspace_size =
          std::max(workspace_size, search1::GetWorkspaceSize(args1, data_algo));
    }

    if (filter_grad) {
      // ------------------- cudnn descriptors ---------------------
      filter_grad_data = filter_grad->data<T>();
      args2.handle = handle;
      args2.idesc.set(transformed_input, iwo_groups);
      args2.wdesc.set(*filter_grad, layout_tensor, iwo_groups);
      args2.odesc.set(transformed_output_grad_channel, iwo_groups);
      args2.cdesc.set(dtype, padding_common, strides, dilations, c_groups);

      using search2 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
      filter_algo =
          search2::Find<T>(args2, exhaustive_search, deterministic, 1, ctx);
      workspace_size = std::max(workspace_size,
                                search2::GetWorkspaceSize(args2, filter_algo));
    }

    // ------------------- cudnn conv backward data ---------------------
    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    if (input_grad) {
      // Because beta is zero, it is unnecessary to reset input_grad.
      for (int i = 0; i < groups; i++) {
        workspace_handle.RunFunc(
            [&](void* cudnn_workspace_ptr) {
              CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionBackwardData(
                  handle, &alpha, args1.wdesc.desc(),
                  filter_data + i * group_offset_filter, args1.odesc.desc(),
                  output_grad_data + i * group_offset_out, args1.cdesc.desc(),
                  data_algo, cudnn_workspace_ptr, workspace_size, &beta,
                  args1.idesc.desc(),
                  transformed_input_grad_data + i * group_offset_in));
            },
            workspace_size);
      }

      std::vector<int> starts(transformed_input_channel.dims().size(), 0);
      std::vector<int> axes(transformed_input_channel.dims().size(), 0);

      for (size_t i = 0; i < transformed_input_channel.dims().size(); ++i) {
        starts[i] = input_pad[2 * i];
        axes[i] = i;
      }

      transformed_input_grad_channel.mutable_data(ctx.GetPlace());
      if (transformed_input_channel.dims().size() == 4) {
        Slice_2<paddle::platform::CUDADeviceContext, T, 4>(
            ctx, &transformed_input_grad, &transformed_input_grad_channel,
            starts, axes);
      } else {
        Slice_2<paddle::platform::CUDADeviceContext, T, 5>(
            ctx, &transformed_input_grad, &transformed_input_grad_channel,
            starts, axes);
      }

      if (channel_last) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_input_grad_channel, input_grad);
      }
    }
    // ------------------- cudnn conv backward filter ---------------------
    if (filter_grad) {
      // Because beta is zero, it is unnecessary to reset filter_grad.
      for (int i = 0; i < groups; i++) {
        workspace_handle.RunFunc(
            [&](void* cudnn_workspace_ptr) {
              CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionBackwardFilter(
                  handle, &alpha, args2.idesc.desc(),
                  input_data + i * group_offset_in, args2.odesc.desc(),
                  output_grad_data + i * group_offset_out, args2.cdesc.desc(),
                  filter_algo, cudnn_workspace_ptr, workspace_size, &beta,
                  args2.wdesc.desc(),
                  filter_grad_data + i * group_offset_filter));
            },
            workspace_size);
      }
    }
  }
};

/*
 * Inputs:  I, W, dO, ddI, ddW
 * Outputs: ddO, dW, dI
 * ddo = conv(ddI, W) + conv(I, ddW)
 * dW = conv_bp_filter(ddI, dO)
 * dI = conv_bp_data(ddW, dO)
 */
template <typename T>
class CUDNNConvDoubleGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "It must use CUDAPlace.");
    auto X = ctx.Input<Tensor>("Input");
    auto W = ctx.Input<Tensor>("Filter");
    auto dO = ctx.Input<Tensor>("DOutput");
    auto ddX = ctx.Input<Tensor>("DDInput");
    auto ddW = ctx.Input<Tensor>("DDFilter");

    auto ddO = ctx.Output<Tensor>("DDOutput");
    auto dW = ctx.Output<Tensor>("DFilter");
    auto dX = ctx.Output<Tensor>("DInput");
    if (ddO) {
      ddO->mutable_data<T>(ctx.GetPlace());
    }
    if (dW) {
      dW->mutable_data<T>(ctx.GetPlace());
    }
    if (dX) {
      dX->mutable_data<T>(ctx.GetPlace());
    }

    // const T* x = X->data<T>();
    const T* dy = dO->data<T>();
    const T* w = W->data<T>();

    const T* ddx = nullptr;
    const T* ddw = nullptr;
    T *dw, *dx, *ddy;
    dw = dx = ddy = nullptr;
    T* transformed_dx = nullptr;
    const std::vector<int>& strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");
    bool deterministic = FLAGS_cudnn_deterministic;
    if (exhaustive_search && deterministic) {
      PADDLE_THROW(
          "Can't set exhaustive_search True and "
          "FLAGS_cudnn_deterministic True at same time.");
    }
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");

    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    // transform Tensors to channel first-----------
    Tensor transformed_X_channel(X->type());
    Tensor transformed_dO_channel(dO->type());
    Tensor transformed_ddX_channel(ddX->type());

    Tensor transformed_ddO_channel(dO->type());
    Tensor transformed_dX_channel(X->type());

    if (channel_last) {
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, X, &transformed_X_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, X, &transformed_X_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, dO, &transformed_dO_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, dO, &transformed_dO_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, ddX, &transformed_ddX_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, ddX, &transformed_ddX_channel);

      if (ddO) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddO, &transformed_ddO_channel);
      }
      if (dX) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, dX, &transformed_dX_channel);
        transformed_dX_channel.mutable_data<T>(ctx.GetPlace());
      }

    } else {
      transformed_X_channel = *X;
      transformed_dO_channel = *dO;
      transformed_ddX_channel = *ddX;
      if (ddO) {
        transformed_ddO_channel.ShareDataWith(*ddO);
      }
      if (dX) {
        transformed_dX_channel.ShareDataWith(*dX);
      }
    }

    auto in_dims = transformed_X_channel.dims();
    auto filter_dims = W->dims();
    framework::DDim in_data_dims =
        framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);
    Tensor transformed_X(X->type());
    Tensor transformed_ddX(X->type());

    Tensor transformed_dX(X->type());

    std::vector<int> padding_common(data_dim, 0);
    std::vector<int> input_pad(X->dims().size() * 2, 0);

    if (!is_sys_pad) {
      // get pad
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_X_channel.dims()[0];
      new_input_shape_vec[1] = transformed_X_channel.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            transformed_X_channel.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_X.Resize(new_input_shape);
      transformed_ddX.Resize(new_input_shape);
      transformed_dX.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_X =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      transformed_ddX =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      if (dX) {
        transformed_dX =
            ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
                new_input_shape, dev_ctx);
      }

      // pad for input
      const int rank = X->dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_ddX_channel, pad_value,
              &transformed_ddX);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_ddX_channel, pad_value,
              &transformed_ddX);
        } break;
        default:
          PADDLE_THROW("ConvOp only support tensors with 4 or 5 dimensions.");
      }

    } else {
      transformed_X.ShareDataWith(transformed_X_channel);
      transformed_ddX.ShareDataWith(transformed_ddX_channel);
      if (dX) {
        transformed_dX.ShareDataWith(transformed_dX_channel);
      }

      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* x = transformed_X.data<T>();

    int iwo_group = groups;
    int c_group = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_group = 1;
    c_group = groups;
#endif
    auto dtype = platform::CudnnDataType<T>::type;

    auto handle = dev_ctx.cudnn_handle();

    ConvArgs args1{&transformed_ddX,         W,
                   &transformed_ddO_channel, strides,
                   padding_common,           dilations};
    ConvArgs args2{&transformed_X, ddW,      &transformed_ddO_channel, strides,
                   padding_common, dilations};
    ConvArgs args3{&transformed_ddX, dW,       &transformed_dO_channel, strides,
                   padding_common,   dilations};
    ConvArgs args4{&transformed_dX, ddW,      &transformed_dO_channel, strides,
                   padding_common,  dilations};

    hipdnnConvolutionFwdAlgo_t fwd_algo1 =
        static_cast<hipdnnConvolutionFwdAlgo_t>(0);
    hipdnnConvolutionFwdAlgo_t fwd_algo2 =
        static_cast<hipdnnConvolutionFwdAlgo_t>(0);
    hipdnnConvolutionBwdDataAlgo_t data_algo =
        static_cast<hipdnnConvolutionBwdDataAlgo_t>(0);
    hipdnnConvolutionBwdFilterAlgo_t filter_algo =
        static_cast<hipdnnConvolutionBwdFilterAlgo_t>(0);

    auto layout = GetCudnnTensorFormat(DataLayout::kNCHW);

    // ddo = conv(ddI, W) + conv(I, ddW)
    size_t workspace_size = 0;

    T* transformed_ddy_channel = nullptr;
    if (ddO) {
      ddy = ddO->data<T>();
      transformed_ddy_channel = transformed_ddO_channel.data<T>();
      if (ddX) {
        args1.handle = handle;
        args1.idesc.set(transformed_ddX, iwo_group);
        args1.wdesc.set(*W, layout, iwo_group);
        args1.odesc.set(transformed_ddO_channel, iwo_group);
        args1.cdesc.set(dtype, padding_common, strides, dilations, c_group);

        using search1 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
        fwd_algo1 = search1::Find<T>(args1, exhaustive_search, false, 0, ctx);
        workspace_size = search1::GetWorkspaceSize(args1, fwd_algo1);
      }

      if (ddW) {
        ddw = ddW->data<T>();
        args2.handle = handle;
        args2.idesc.set(transformed_X, iwo_group);

        args2.wdesc.set(*ddW, layout, iwo_group);

        args2.odesc.set(transformed_ddO_channel, iwo_group);
        args2.cdesc.set(dtype, padding_common, strides, dilations, c_group);

        using search2 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
        fwd_algo2 = search2::Find<T>(args2, exhaustive_search, false, 0, ctx);
        workspace_size = std::max(workspace_size,
                                  search2::GetWorkspaceSize(args2, fwd_algo2));
      }
    }

    if (dW && ddX) {
      dw = dW->data<T>();
      args3.handle = handle;
      args3.idesc.set(transformed_ddX, iwo_group);
      args3.wdesc.set(*dW, layout, iwo_group);

      args3.odesc.set(transformed_dO_channel, iwo_group);

      args3.cdesc.set(dtype, padding_common, strides, dilations, c_group);

      using search3 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
      filter_algo =
          search3::Find<T>(args3, exhaustive_search, deterministic, 1, ctx);
      workspace_size = std::max(workspace_size,
                                search3::GetWorkspaceSize(args3, filter_algo));
    }

    if (ddW && dX) {
      transformed_dx = transformed_dX.data<T>();

      args4.handle = handle;
      args4.idesc.set(transformed_dX, iwo_group);
      args4.wdesc.set(*ddW, layout, iwo_group);
      args4.odesc.set(transformed_dO_channel, iwo_group);
      args4.cdesc.set(dtype, padding_common, strides, dilations, c_group);

      using search4 = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
      data_algo =
          search4::Find<T>(args4, exhaustive_search, deterministic, 2, ctx);
      workspace_size =
          std::max(workspace_size, search4::GetWorkspaceSize(args4, data_algo));
    }

    int i_n, i_c, i_d, i_h, i_w;
    GetNCDHW(transformed_X.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d, &i_h,
             &i_w);

    int o_n, o_c, o_d, o_h, o_w;
    GetNCDHW(transformed_dO_channel.dims(), DataLayout::kNCHW, &o_n, &o_c, &o_d,
             &o_h, &o_w);

    int group_offset_in = i_c / groups * i_h * i_w * i_d;
    int group_offset_out = o_c / groups * o_h * o_w * o_d;
    int group_offset_filter = W->numel() / groups;

    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    auto wkspace_handle = dev_ctx.cudnn_workspace_handle();

    if (ddO) {
      if (ddX) {
        ddx = transformed_ddX.data<T>();
        for (int i = 0; i < groups; i++) {
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionForward(
                    handle, &alpha, args1.idesc.desc(),
                    ddx + i * group_offset_in, args1.wdesc.desc(),
                    w + i * group_offset_filter, args1.cdesc.desc(), fwd_algo1,
                    workspace_ptr, workspace_size, &beta, args1.odesc.desc(),
                    transformed_ddy_channel + i * group_offset_out));
              },
              workspace_size);
        }
      }
      if (ddW) {
        for (int i = 0; i < groups; i++) {
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionForward(
                    handle, &alpha, args2.idesc.desc(), x + i * group_offset_in,
                    args2.wdesc.desc(), ddw + i * group_offset_filter,
                    args2.cdesc.desc(), fwd_algo2, workspace_ptr,
                    workspace_size, &alpha, args2.odesc.desc(),
                    transformed_ddy_channel + i * group_offset_out));
              },
              workspace_size);
        }
      }
      if (channel_last) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_ddO_channel, ddO);
      }
    }
    T* transformed_dy_channel = nullptr;
    if (dW && ddX) {
      ddx = transformed_ddX.data<T>();
      transformed_dy_channel = transformed_dO_channel.data<T>();
      for (int i = 0; i < groups; i++) {
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionBackwardFilter(
                  handle, &alpha, args3.idesc.desc(), ddx + i * group_offset_in,
                  args3.odesc.desc(),
                  transformed_dy_channel + i * group_offset_out,
                  args3.cdesc.desc(), filter_algo, workspace_ptr,
                  workspace_size, &beta, args3.wdesc.desc(),
                  dw + i * group_offset_filter));
            },
            workspace_size);
      }
    }

    if (dX && ddW) {
      ddw = ddW->data<T>();
      for (int i = 0; i < groups; i++) {
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionBackwardData(
                  handle, &alpha, args4.wdesc.desc(),
                  ddw + i * group_offset_filter, args4.odesc.desc(),
                  transformed_dy_channel + i * group_offset_out,
                  args4.cdesc.desc(), data_algo, workspace_ptr, workspace_size,
                  &beta, args4.idesc.desc(),
                  transformed_dx + i * group_offset_in));
            },
            workspace_size);
      }

      // reverse padded input
      std::vector<int> starts(X->dims().size(), 0);
      std::vector<int> axes(X->dims().size(), 0);

      for (size_t i = 0; i < X->dims().size(); ++i) {
        starts[i] = input_pad[2 * i];
        axes[i] = i;
      }
      if (X->dims().size() == 4) {
        Slice_2<paddle::platform::CUDADeviceContext, T, 4>(
            ctx, &transformed_dX, &transformed_dX_channel, starts, axes);
      } else {
        Slice_2<paddle::platform::CUDADeviceContext, T, 5>(
            ctx, &transformed_dX, &transformed_dX_channel, starts, axes);
      }
      if (channel_last) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_dX_channel, dX);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace plat = paddle::platform;
REGISTER_OP_KERNEL(conv2d, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvOpKernel<float>,
                   paddle::operators::CUDNNConvOpKernel<double>,
                   paddle::operators::CUDNNConvOpKernel<plat::float16>);
REGISTER_OP_KERNEL(conv2d_grad, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvGradOpKernel<float>,
                   paddle::operators::CUDNNConvGradOpKernel<double>,
                   paddle::operators::CUDNNConvGradOpKernel<plat::float16>);
REGISTER_OP_KERNEL(
    conv2d_grad_grad, CUDNN, plat::CUDAPlace,
    paddle::operators::CUDNNConvDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<double>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<plat::float16>);

REGISTER_OP_KERNEL(conv3d, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvOpKernel<float>,
                   paddle::operators::CUDNNConvOpKernel<double>,
                   paddle::operators::CUDNNConvOpKernel<plat::float16>);
REGISTER_OP_KERNEL(conv3d_grad, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvGradOpKernel<float>,
                   paddle::operators::CUDNNConvGradOpKernel<double>);
REGISTER_OP_KERNEL(
    conv3d_grad_grad, CUDNN, plat::CUDAPlace,
    paddle::operators::CUDNNConvDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<double>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<plat::float16>);
