#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the spopecific language governing permissions and
limitations under the License. */

#include <utility>
#include <vector>

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#include "paddle/fluid/operators/conv_cudnn_op_cache.h"
#include "paddle/fluid/operators/conv_op.h"
#include "paddle/fluid/operators/math/padding.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/cudnn_workspace_helper.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/profiler.h"

DECLARE_bool(cudnn_deterministic);
DECLARE_uint64(conv_workspace_size_limit);
DECLARE_bool(cudnn_exhaustive_search);

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using DataLayout = platform::DataLayout;

static inline bool IsVoltaOrLater(const platform::CUDADeviceContext& dev_ctx) {
  return dev_ctx.GetComputeCapability() >= 70;
}

template <typename T>
class CUDNNConvOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        paddle::platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    const Tensor* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* output = ctx.Output<Tensor>("Output");
    output->mutable_data<T>(ctx.GetPlace());
    const std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");

    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");
    bool deterministic = FLAGS_cudnn_deterministic;
    auto exhaustive_deterministic = exhaustive_search && deterministic;
    PADDLE_ENFORCE_EQ(exhaustive_deterministic, false,
                      platform::errors::InvalidArgument(
                          "Cann't set exhaustive_search True and "
                          "FLAGS_cudnn_deterministic True at same time."));

    const std::string padding_algorithm =
        ctx.Attr<std::string>("padding_algorithm");
    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    auto dtype = platform::CudnnDataType<T>::type;

    // Tensor Core introduced from Volta GPUs supports more faster conv op
    // with FP16 in NHWC data format.
    const bool compute_in_nhwc =
        dtype == HIPDNN_DATA_HALF && IsVoltaOrLater(dev_ctx);
    // We will only do data format conversion from NHWC to NCHW.
    // cudnn will convert NCHW to NHWC automatically on Tensor Core.
    auto compute_format =
        compute_in_nhwc && channel_last ? DataLayout::kNHWC : DataLayout::kNCHW;
    VLOG(3) << "Compute ConvOp with cuDNN:"
            << " data_format=" << data_format << " compute_format="
            << (compute_format == DataLayout::kNHWC ? "NHWC" : "NCHW");

    // ------------ transformed tensor -----------
    Tensor transformed_input_channel(input->type());
    Tensor transformed_output(output->type());
    Tensor transformed_filter_channel(filter->type());
    T* output_data = nullptr;
    if (channel_last && compute_format == DataLayout::kNCHW) {
      VLOG(3) << "Transform input tensor from NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, output,
                                                           &transformed_output);

    } else {
      transformed_input_channel.ShareDataWith(*input);
      transformed_output.ShareDataWith(*output);
    }
    if (compute_format == DataLayout::kNHWC) {
      VLOG(3) << "Transform filter tensor from NCHW to NHWC.";
      ResizeToChannelLast<platform::CUDADeviceContext, T>(
          ctx, filter, &transformed_filter_channel);
      TransToChannelLast<platform::CUDADeviceContext, T>(
          ctx, filter, &transformed_filter_channel);
    } else {
      transformed_filter_channel.ShareDataWith(*filter);
    }
    output_data = transformed_output.data<T>();

    // update padding and dilation
    auto in_dims = transformed_input_channel.dims();
    auto filter_dims = transformed_filter_channel.dims();
    framework::DDim in_data_dims;
    framework::DDim filter_data_dims;

    if (compute_format == DataLayout::kNCHW) {
      in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
      filter_data_dims =
          framework::slice_ddim(filter_dims, 2, filter_dims.size());
    } else {
      in_data_dims = framework::slice_ddim(in_dims, 1, in_dims.size() - 1);
      filter_data_dims =
          framework::slice_ddim(filter_dims, 1, filter_dims.size() - 1);
    }

    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    Tensor transformed_input;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_input_channel.dims()[0];

      if (compute_format == DataLayout::kNCHW) {
        new_input_shape_vec[1] = transformed_input_channel.dims()[1];
      } else {
        new_input_shape_vec[data_dim + 1] =
            transformed_input_channel.dims()[data_dim + 1];
      }

      std::vector<int> input_pad(transformed_input_channel.dims().size() * 2,
                                 0);
      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        if (compute_format == DataLayout::kNCHW) {
          new_input_shape_vec[i + 2] =
              transformed_input_channel.dims()[i + 2] + padding_diff[i];
        } else {
          new_input_shape_vec[i + 1] =
              transformed_input_channel.dims()[i + 1] + padding_diff[i];
        }
        if (compute_format == DataLayout::kNCHW) {
          input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
          input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
        } else {
          input_pad[2 * i + 2] = paddings[2 * i] - padding_common[i];
          input_pad[2 * i + 2 + 1] = paddings[2 * i + 1] - padding_common[i];
        }
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      const int rank = transformed_input_channel.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        default:
          PADDLE_THROW(platform::errors::InvalidArgument(
              "ConvOp only support tensors with 4 or 5 dimensions."));
      }

    } else {
      transformed_input.ShareDataWith(transformed_input_channel);
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = transformed_input.data<T>();
    const T* filter_data = transformed_filter_channel.data<T>();

    // ------------------- cudnn descriptors ---------------------
    ConvArgs args{&transformed_input,
                  &transformed_filter_channel,
                  &transformed_output,
                  strides,
                  padding_common,
                  dilations,
                  dtype};

    auto handle = dev_ctx.cudnn_handle();
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    DataLayout layout = compute_format == DataLayout::kNHWC ? DataLayout::kNHWC
                                                            : DataLayout::kNCHW;
    if (transformed_input.dims().size() == 5) {
      layout = compute_format == DataLayout::kNHWC ? DataLayout::kNDHWC
                                                   : DataLayout::kNCDHW;
    }
    auto layout_format = GetCudnnTensorFormat(layout);

    args.handle = handle;
    args.cdesc.set(dtype, padding_common, strides, dilations,
                   platform::AllowTF32Cudnn());

#if CUDNN_VERSION_MIN(7, 0, 1)
    // cudnn 7 can support groups, no need to do it manually
    // FIXME(typhoonzero): find a better way to disable groups
    // rather than setting it to 1.
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnSetConvolutionGroupCount(args.cdesc.desc(),
                                                         groups));
    groups = 1;
#endif
    args.idesc.set(transformed_input, layout_format);
    args.wdesc.set(transformed_filter_channel, layout_format, groups);
    args.odesc.set(transformed_output, layout_format);
    int i_n, i_c, i_d, i_h, i_w;
    int o_n, o_c, o_d, o_h, o_w;

    if (compute_format == DataLayout::kNHWC) {
      GetNCDHW(transformed_input.dims(), DataLayout::kNHWC, &i_n, &i_c, &i_d,
               &i_h, &i_w);
      GetNCDHW(transformed_output.dims(), DataLayout::kNHWC, &o_n, &o_c, &o_d,
               &o_h, &o_w);
    } else {
      GetNCDHW(transformed_input.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d,
               &i_h, &i_w);
      GetNCDHW(transformed_output.dims(), DataLayout::kNCHW, &o_n, &o_c, &o_d,
               &o_h, &o_w);
    }

    int group_offset_in = i_c / groups * i_h * i_w * i_d;
    int group_offset_out = o_c / groups * o_h * o_w * o_d;
    int group_offset_filter = transformed_filter_channel.numel() / groups;
    // ------------------- cudnn conv workspace ---------------------
    size_t workspace_size = 0;  // final workspace to allocate.
    // ------------------- cudnn conv algorithm ---------------------
    hipdnnConvolutionFwdAlgo_t algo{};

    using search = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
    algo = search::Find<T>(args, exhaustive_search, false, ctx);
    workspace_size = search::GetWorkspaceSize(args, algo);

#if CUDNN_VERSION_MIN(7, 0, 1)
    // when groups > 1, SearchAlgorithm find algo is CUDNN_CONVOLUTION_\
    // FWD_ALGO_WINOGRAD_NONFUSED, but this kind of algorithm is unstable
    // in forward computation, so change the algorithm to CUDNN_CONVOLUTION_\
    // FWD_ALGO_IMPLICIT_GEMM manually.
    if (ctx.Attr<int>("groups") > 1) {
      algo = static_cast<hipdnnConvolutionFwdAlgo_t>(0);
    }
#endif

    // ------------------- cudnn conv forward ---------------------
    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = 0.0f;

    // NOTE(zhiqiu): inplace addto is not supportted in double grad yet.
    // ScalingParamType<T> beta = ctx.Attr<bool>("use_addto") ? 1.0f : 0.0f;
    // VLOG(4) << "Conv: use_addto = " << ctx.Attr<bool>("use_addto");

    for (int i = 0; i < groups; i++) {
      workspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_CUDA_SUCCESS(
                platform::dynload::hipdnnConvolutionForward(
                    handle, &alpha, args.idesc.desc(),
                    input_data + i * group_offset_in, args.wdesc.desc(),
                    filter_data + i * group_offset_filter, args.cdesc.desc(),
                    algo, workspace_ptr, workspace_size, &beta,
                    args.odesc.desc(), output_data + i * group_offset_out));
          },
          workspace_size);
    }

    if (channel_last && compute_format == DataLayout::kNCHW) {
      TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
          ctx, &transformed_output, output);
    }
  }
};

template <typename T>
class CUDNNConvGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        paddle::platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    auto input = ctx.Input<Tensor>("Input");
    auto filter = ctx.Input<Tensor>("Filter");
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));

    if (input_grad) {
      input_grad->mutable_data<T>(ctx.GetPlace());
    }
    if (filter_grad) {
      filter_grad->mutable_data<T>(ctx.GetPlace());
    }

    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    int groups = ctx.Attr<int>("groups");

    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");
    bool deterministic = FLAGS_cudnn_deterministic;
    auto exhaustive_deterministic = exhaustive_search && deterministic;
    PADDLE_ENFORCE_EQ(exhaustive_deterministic, false,
                      platform::errors::InvalidArgument(
                          "Cann't set exhaustive_search True and "
                          "FLAGS_cudnn_deterministic True at same time."));

    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    auto dtype = platform::CudnnDataType<T>::type;
    const bool compute_in_nhwc =
        dtype == HIPDNN_DATA_HALF && IsVoltaOrLater(dev_ctx);
    auto compute_format =
        compute_in_nhwc && channel_last ? DataLayout::kNHWC : DataLayout::kNCHW;
    VLOG(3) << "Compute ConvGradOp with cuDNN:"
            << " data_format=" << data_format << " compute_format="
            << (compute_format == DataLayout::kNHWC ? "NHWC" : "NCHW");

    // transform Tensor
    Tensor transformed_input_channel(input->type());
    Tensor transformed_output_grad_channel(output_grad->type());
    Tensor transformed_input_grad_channel(input->type());
    Tensor transformed_filter_channel(filter->type());
    Tensor transformed_filter_grad_channel(filter->type());

    if (channel_last && compute_format == DataLayout::kNCHW) {
      VLOG(3) << "Transform input, output_grad, input_grad and tensor from "
                 "NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, input, &transformed_input_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, output_grad, &transformed_output_grad_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, output_grad, &transformed_output_grad_channel);

      if (input_grad) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, input_grad, &transformed_input_grad_channel);
        // NOTE(zhiqiu): If inplace_addto strategy is enabled, we need to copy
        // the data of input_grad to transformed_input_grad_channel.
        if (ctx.Attr<bool>("use_addto")) {
          TransToChannelFirst<platform::CUDADeviceContext, T>(
              ctx, input_grad, &transformed_input_grad_channel);
        }
      }
    } else {
      transformed_input_channel.ShareDataWith(*input);
      transformed_output_grad_channel.ShareDataWith(*output_grad);
      if (input_grad) {
        transformed_input_grad_channel.ShareDataWith(*input_grad);
      }
    }

    if (compute_format == DataLayout::kNHWC) {
      VLOG(3) << "Transform filter and filter_grad tensor from NCHW to NHWC.";
      ResizeToChannelLast<platform::CUDADeviceContext, T>(
          ctx, filter, &transformed_filter_channel);
      TransToChannelLast<platform::CUDADeviceContext, T>(
          ctx, filter, &transformed_filter_channel);

      if (filter_grad) {
        ResizeToChannelLast<platform::CUDADeviceContext, T>(
            ctx, filter_grad, &transformed_filter_grad_channel);
      }
    } else {
      transformed_filter_channel.ShareDataWith(*filter);
      if (filter_grad) {
        transformed_filter_grad_channel.ShareDataWith(*filter_grad);
      }
    }

    //  update paddings
    auto in_dims = transformed_input_channel.dims();
    auto filter_dims = transformed_filter_channel.dims();
    framework::DDim in_data_dims;
    framework::DDim filter_data_dims;
    if (compute_format == DataLayout::kNCHW) {
      in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
      filter_data_dims =
          framework::slice_ddim(filter_dims, 2, filter_dims.size());
    } else {
      in_data_dims = framework::slice_ddim(in_dims, 1, in_dims.size() - 1);
      filter_data_dims =
          framework::slice_ddim(filter_dims, 1, filter_dims.size() - 1);
    }
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    // cuDNN only supports padding the same amount on every dimension.
    // So we create a new padded input tensor.
    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);
    Tensor transformed_input(input->type());
    Tensor transformed_input_grad(input->type());
    std::vector<int> padding_common(data_dim, 0);
    std::vector<int> input_pad(transformed_input_channel.dims().size() * 2, 0);

    if (!is_sys_pad) {
      // get pad
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_input_channel.dims()[0];
      if (compute_format == DataLayout::kNCHW) {
        new_input_shape_vec[1] = transformed_input_channel.dims()[1];
      } else {
        new_input_shape_vec[data_dim + 1] =
            transformed_input_channel.dims()[data_dim + 1];
      }

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        if (compute_format == DataLayout::kNCHW) {
          new_input_shape_vec[i + 2] =
              transformed_input_channel.dims()[i + 2] + padding_diff[i];
        } else {
          new_input_shape_vec[i + 1] =
              transformed_input_channel.dims()[i + 1] + padding_diff[i];
        }
        if (compute_format == DataLayout::kNCHW) {
          input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
          input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
        } else {
          input_pad[2 * i + 2] = paddings[2 * i] - padding_common[i];
          input_pad[2 * i + 2 + 1] = paddings[2 * i + 1] - padding_common[i];
        }
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);

      transformed_input_grad.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      if (input_grad) {
        transformed_input_grad =
            ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
                new_input_shape, dev_ctx);
      }
      // pad for input
      const int rank = transformed_input_channel.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        default:
          PADDLE_THROW(platform::errors::InvalidArgument(
              "ConvOp only support tensors with 4 or 5 dimensions."));
      }
    } else {
      transformed_input.ShareDataWith(transformed_input_channel);
      if (input_grad) {
        transformed_input_grad.ShareDataWith(transformed_input_grad_channel);
      }
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = transformed_input.data<T>();
    const T* output_grad_data = transformed_output_grad_channel.data<T>();
    const T* filter_data = transformed_filter_channel.data<T>();
    T* filter_grad_data = nullptr;
    T* input_grad_data = nullptr;
    T* transformed_input_grad_data = nullptr;

    ConvArgs args1{&transformed_input_grad,
                   &transformed_filter_channel,
                   &transformed_output_grad_channel,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args2{&transformed_input,
                   &transformed_filter_grad_channel,
                   &transformed_output_grad_channel,
                   strides,
                   padding_common,
                   dilations,
                   dtype};

    auto handle = dev_ctx.cudnn_handle();
    DataLayout layout = compute_format == DataLayout::kNHWC ? DataLayout::kNHWC
                                                            : DataLayout::kNCHW;
    if (transformed_input.dims().size() == 5) {
      layout = compute_format == DataLayout::kNHWC ? DataLayout::kNDHWC
                                                   : DataLayout::kNCDHW;
    }
    auto layout_tensor = GetCudnnTensorFormat(layout);
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();

    int i_n, i_c, i_d, i_h, i_w;
    int o_n, o_c, o_d, o_h, o_w;
    if (compute_format == DataLayout::kNHWC) {
      GetNCDHW(transformed_input.dims(), DataLayout::kNHWC, &i_n, &i_c, &i_d,
               &i_h, &i_w);
      GetNCDHW(transformed_output_grad_channel.dims(), DataLayout::kNHWC, &o_n,
               &o_c, &o_d, &o_h, &o_w);
    } else {
      GetNCDHW(transformed_input.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d,
               &i_h, &i_w);
      GetNCDHW(transformed_output_grad_channel.dims(), DataLayout::kNCHW, &o_n,
               &o_c, &o_d, &o_h, &o_w);
    }

    int group_offset_in = i_c / groups * i_h * i_w * i_d;
    int group_offset_out = o_c / groups * o_h * o_w * o_d;
    int group_offset_filter = transformed_filter_channel.numel() / groups;
    // ------------------- cudnn backward algorithm ---------------------
    hipdnnConvolutionBwdDataAlgo_t data_algo =
        static_cast<hipdnnConvolutionBwdDataAlgo_t>(0);
    hipdnnConvolutionBwdFilterAlgo_t filter_algo =
        static_cast<hipdnnConvolutionBwdFilterAlgo_t>(0);
    size_t workspace_size = 0;
    int iwo_groups = groups;
    int c_groups = 1;

#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_groups = 1;
    c_groups = groups;
    groups = 1;
#endif

    if (input_grad) {
      // ------------------- cudnn descriptors ---------------------
      input_grad_data = input_grad->data<T>();
      transformed_input_grad_data = transformed_input_grad.data<T>();
      args1.handle = handle;
      args1.idesc.set(transformed_input_grad, layout_tensor);
      args1.wdesc.set(transformed_filter_channel, layout_tensor, iwo_groups);
      args1.odesc.set(transformed_output_grad_channel, layout_tensor);
      args1.cdesc.set(dtype, padding_common, strides, dilations,
                      platform::AllowTF32Cudnn(), c_groups);

      using search1 = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
      data_algo =
          search1::Find<T>(args1, exhaustive_search, deterministic, ctx);
      workspace_size =
          std::max(workspace_size, search1::GetWorkspaceSize(args1, data_algo));
    }

    if (filter_grad) {
      // ------------------- cudnn descriptors ---------------------
      filter_grad_data = transformed_filter_grad_channel.data<T>();
      args2.handle = handle;
      args2.idesc.set(transformed_input, layout_tensor);
      args2.wdesc.set(transformed_filter_grad_channel, layout_tensor,
                      iwo_groups);
      args2.odesc.set(transformed_output_grad_channel, layout_tensor);
      args2.cdesc.set(dtype, padding_common, strides, dilations,
                      platform::AllowTF32Cudnn(), c_groups);

      using search2 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
      filter_algo =
          search2::Find<T>(args2, exhaustive_search, deterministic, ctx);
      workspace_size = std::max(workspace_size,
                                search2::GetWorkspaceSize(args2, filter_algo));
    }

    // ------------------- cudnn conv backward data ---------------------
    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = ctx.Attr<bool>("use_addto") ? 1.0f : 0.0f;
    VLOG(4) << "Conv_grad: use_addto = " << ctx.Attr<bool>("use_addto");

    if (input_grad) {
      // When beta is 0, it is unnecessary to reset input_grad.
      // When beta is 1, the output cannot be reset since addt strategy used.
      for (int i = 0; i < groups; i++) {
        workspace_handle.RunFunc(
            [&](void* cudnn_workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::hipdnnConvolutionBackwardData(
                      handle, &alpha, args1.wdesc.desc(),
                      filter_data + i * group_offset_filter, args1.odesc.desc(),
                      output_grad_data + i * group_offset_out,
                      args1.cdesc.desc(), data_algo, cudnn_workspace_ptr,
                      workspace_size, &beta, args1.idesc.desc(),
                      transformed_input_grad_data + i * group_offset_in));
            },
            workspace_size);
      }

      if (!is_sys_pad) {
        std::vector<int> starts(transformed_input_channel.dims().size(), 0);
        std::vector<int> axes(transformed_input_channel.dims().size(), 0);

        for (size_t i = 0; i < transformed_input_channel.dims().size(); ++i) {
          starts[i] = input_pad[2 * i];
          axes[i] = i;
        }

        transformed_input_grad_channel.mutable_data(ctx.GetPlace());
        if (transformed_input_channel.dims().size() == 4) {
          RemovePaddingSlice<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, &transformed_input_grad, &transformed_input_grad_channel,
              starts, axes);
        } else {
          RemovePaddingSlice<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, &transformed_input_grad, &transformed_input_grad_channel,
              starts, axes);
        }
      }

      if (channel_last && compute_format == DataLayout::kNCHW) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_input_grad_channel, input_grad);
      }
    }

    // filter_grad do not use inplace addto.
    ScalingParamType<T> beta_filter = 0.0f;
    // ------------------- cudnn conv backward filter ---------------------
    if (filter_grad) {
      // Because beta is zero, it is unnecessary to reset filter_grad.
      for (int i = 0; i < groups; i++) {
        workspace_handle.RunFunc(
            [&](void* cudnn_workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::hipdnnConvolutionBackwardFilter(
                      handle, &alpha, args2.idesc.desc(),
                      input_data + i * group_offset_in, args2.odesc.desc(),
                      output_grad_data + i * group_offset_out,
                      args2.cdesc.desc(), filter_algo, cudnn_workspace_ptr,
                      workspace_size, &beta_filter, args2.wdesc.desc(),
                      filter_grad_data + i * group_offset_filter));
            },
            workspace_size);
      }

      if (compute_format == DataLayout::kNHWC) {
        TransToChannelFirst<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_filter_grad_channel, filter_grad);
      }
    }
  }
};

/*
 * Inputs:  I, W, dO, ddI, ddW
 * Outputs: ddO, dW, dI
 * ddo = conv(ddI, W) + conv(I, ddW)
 * dW = conv_bp_filter(ddI, dO)
 * dI = conv_bp_data(ddW, dO)
 */
template <typename T>
class CUDNNConvDoubleGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        paddle::platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    auto X = ctx.Input<Tensor>("Input");
    auto W = ctx.Input<Tensor>("Filter");
    auto dO = ctx.Input<Tensor>("DOutput");
    auto ddX = ctx.Input<Tensor>("DDInput");
    auto ddW = ctx.Input<Tensor>("DDFilter");

    auto ddO = ctx.Output<Tensor>("DDOutput");
    auto dW = ctx.Output<Tensor>("DFilter");
    auto dX = ctx.Output<Tensor>("DInput");
    if (ddO) {
      ddO->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<platform::CUDADeviceContext, T> set_zero;
      set_zero(dev_ctx, ddO, static_cast<T>(0));
    }
    if (dW) {
      dW->mutable_data<T>(ctx.GetPlace());
    }
    if (dX) {
      dX->mutable_data<T>(ctx.GetPlace());
    }

    // const T* x = X->data<T>();
    const T* dy = dO->data<T>();
    const T* w = W->data<T>();

    const T* ddx = nullptr;
    const T* ddw = nullptr;
    T *dw, *dx, *ddy;
    dw = dx = ddy = nullptr;
    T* transformed_dx = nullptr;
    const std::vector<int>& strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");

    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");
    bool deterministic = FLAGS_cudnn_deterministic;
    auto exhaustive_deterministic = exhaustive_search && deterministic;
    PADDLE_ENFORCE_EQ(exhaustive_deterministic, false,
                      platform::errors::InvalidArgument(
                          "Cann't set exhaustive_search True and "
                          "FLAGS_cudnn_deterministic True at same time."));

    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");

    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    // transform Tensors to channel first-----------
    Tensor transformed_X_channel(X->type());
    Tensor transformed_dO_channel(dO->type());
    Tensor transformed_ddX_channel(X->type());

    Tensor transformed_ddO_channel(dO->type());
    Tensor transformed_dX_channel(X->type());

    if (channel_last) {
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, X, &transformed_X_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, X, &transformed_X_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, dO, &transformed_dO_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, dO, &transformed_dO_channel);

      if (ddX) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddX, &transformed_ddX_channel);
        TransToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddX, &transformed_ddX_channel);
      }

      if (ddO) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddO, &transformed_ddO_channel);
      }
      if (dX) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, dX, &transformed_dX_channel);
        transformed_dX_channel.mutable_data<T>(ctx.GetPlace());
      }

    } else {
      transformed_X_channel = *X;
      transformed_dO_channel = *dO;
      if (ddX) {
        transformed_ddX_channel = *ddX;
      }
      if (ddO) {
        transformed_ddO_channel.ShareDataWith(*ddO);
      }
      if (dX) {
        transformed_dX_channel.ShareDataWith(*dX);
      }
    }

    auto in_dims = transformed_X_channel.dims();
    auto filter_dims = W->dims();
    framework::DDim in_data_dims =
        framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);
    Tensor transformed_X(X->type());
    Tensor transformed_ddX(X->type());

    Tensor transformed_dX(X->type());

    std::vector<int> padding_common(data_dim, 0);
    std::vector<int> input_pad(X->dims().size() * 2, 0);

    if (!is_sys_pad) {
      // get pad
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_X_channel.dims()[0];
      new_input_shape_vec[1] = transformed_X_channel.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            transformed_X_channel.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_X.Resize(new_input_shape);
      transformed_ddX.Resize(new_input_shape);
      transformed_dX.Resize(new_input_shape);

      transformed_X =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      if (ddX) {
        transformed_ddX =
            ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
                new_input_shape, dev_ctx);
      }
      if (dX) {
        transformed_dX =
            ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
                new_input_shape, dev_ctx);
      }

      // pad for input
      const int rank = X->dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
          if (ddX) {
            math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
                ctx, input_pad, transformed_ddX_channel, pad_value,
                &transformed_ddX);
          }
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
          if (ddX) {
            math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
                ctx, input_pad, transformed_ddX_channel, pad_value,
                &transformed_ddX);
          }
        } break;
        default:
          PADDLE_THROW(platform::errors::InvalidArgument(
              "ConvOp only support tensors with 4 or 5 dimensions."));
      }

    } else {
      transformed_X.ShareDataWith(transformed_X_channel);
      if (ddX) {
        transformed_ddX.ShareDataWith(transformed_ddX_channel);
      }
      if (dX) {
        transformed_dX.ShareDataWith(transformed_dX_channel);
      }

      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* x = transformed_X.data<T>();

    int iwo_group = groups;
    int c_group = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_group = 1;
    c_group = groups;
    groups = 1;
#endif
    auto dtype = platform::CudnnDataType<T>::type;

    auto handle = dev_ctx.cudnn_handle();

    ConvArgs args1{&transformed_ddX,
                   W,
                   &transformed_ddO_channel,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args2{
        &transformed_X, ddW,  &transformed_ddO_channel, strides, padding_common,
        dilations,      dtype};
    ConvArgs args3{&transformed_ddX,
                   dW,
                   &transformed_dO_channel,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args4{
        &transformed_dX, ddW,  &transformed_dO_channel, strides, padding_common,
        dilations,       dtype};

    hipdnnConvolutionFwdAlgo_t fwd_algo1 =
        static_cast<hipdnnConvolutionFwdAlgo_t>(0);
    hipdnnConvolutionFwdAlgo_t fwd_algo2 =
        static_cast<hipdnnConvolutionFwdAlgo_t>(0);
    hipdnnConvolutionBwdDataAlgo_t data_algo =
        static_cast<hipdnnConvolutionBwdDataAlgo_t>(0);
    hipdnnConvolutionBwdFilterAlgo_t filter_algo =
        static_cast<hipdnnConvolutionBwdFilterAlgo_t>(0);

    auto layout = GetCudnnTensorFormat(DataLayout::kNCHW);

    // ddo = conv(ddI, W) + conv(I, ddW)
    size_t workspace_size = 0;

    T* transformed_ddy_channel = nullptr;
    if (ddO) {
      ddy = ddO->data<T>();
      transformed_ddy_channel = transformed_ddO_channel.data<T>();
      if (ddX) {
        args1.handle = handle;
        args1.idesc.set(transformed_ddX, iwo_group);
        args1.wdesc.set(*W, layout, iwo_group);
        args1.odesc.set(transformed_ddO_channel, iwo_group);
        args1.cdesc.set(dtype, padding_common, strides, dilations,
                        platform::AllowTF32Cudnn(), c_group);

        using search1 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
        fwd_algo1 = search1::Find<T>(args1, exhaustive_search, false, ctx);
        workspace_size = search1::GetWorkspaceSize(args1, fwd_algo1);
      }

      if (ddW) {
        ddw = ddW->data<T>();
        args2.handle = handle;
        args2.idesc.set(transformed_X, iwo_group);

        args2.wdesc.set(*ddW, layout, iwo_group);

        args2.odesc.set(transformed_ddO_channel, iwo_group);
        args2.cdesc.set(dtype, padding_common, strides, dilations,
                        platform::AllowTF32Cudnn(), c_group);

        using search2 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
        fwd_algo2 = search2::Find<T>(args2, exhaustive_search, false, ctx);
        workspace_size = std::max(workspace_size,
                                  search2::GetWorkspaceSize(args2, fwd_algo2));
      }
    }

    if (dW && ddX) {
      dw = dW->data<T>();
      args3.handle = handle;
      args3.idesc.set(transformed_ddX, iwo_group);
      args3.wdesc.set(*dW, layout, iwo_group);

      args3.odesc.set(transformed_dO_channel, iwo_group);

      args3.cdesc.set(dtype, padding_common, strides, dilations,
                      platform::AllowTF32Cudnn(), c_group);

      using search3 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
      filter_algo =
          search3::Find<T>(args3, exhaustive_search, deterministic, ctx);
      workspace_size = std::max(workspace_size,
                                search3::GetWorkspaceSize(args3, filter_algo));
    }

    if (ddW && dX) {
      transformed_dx = transformed_dX.data<T>();

      args4.handle = handle;
      args4.idesc.set(transformed_dX, iwo_group);
      args4.wdesc.set(*ddW, layout, iwo_group);
      args4.odesc.set(transformed_dO_channel, iwo_group);
      args4.cdesc.set(dtype, padding_common, strides, dilations,
                      platform::AllowTF32Cudnn(), c_group);

      using search4 = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
      data_algo =
          search4::Find<T>(args4, exhaustive_search, deterministic, ctx);
      workspace_size =
          std::max(workspace_size, search4::GetWorkspaceSize(args4, data_algo));
    }

    int i_n, i_c, i_d, i_h, i_w;
    GetNCDHW(transformed_X.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d, &i_h,
             &i_w);

    int o_n, o_c, o_d, o_h, o_w;
    GetNCDHW(transformed_dO_channel.dims(), DataLayout::kNCHW, &o_n, &o_c, &o_d,
             &o_h, &o_w);

    int group_offset_in = i_c / groups * i_h * i_w * i_d;
    int group_offset_out = o_c / groups * o_h * o_w * o_d;
    int group_offset_filter = W->numel() / groups;

    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = 0.0f;

    // NOTE(zhiqiu): inplace addto is not supportted in double grad yet.
    // ScalingParamType<T> beta = ctx.Attr<bool>("use_addto") ? 1.0f :
    // 0.0f;
    // VLOG(4) << "Conv_grad_grad: use_addto = " << ctx.Attr<bool>("use_addto");

    auto wkspace_handle = dev_ctx.cudnn_workspace_handle();

    if (ddO) {
      if (ddX) {
        ddx = transformed_ddX.data<T>();
        for (int i = 0; i < groups; i++) {
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                PADDLE_ENFORCE_CUDA_SUCCESS(
                    platform::dynload::hipdnnConvolutionForward(
                        handle, &alpha, args1.idesc.desc(),
                        ddx + i * group_offset_in, args1.wdesc.desc(),
                        w + i * group_offset_filter, args1.cdesc.desc(),
                        fwd_algo1, workspace_ptr, workspace_size, &beta,
                        args1.odesc.desc(),
                        transformed_ddy_channel + i * group_offset_out));
              },
              workspace_size);
        }
      }
      if (ddW) {
        for (int i = 0; i < groups; i++) {
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                PADDLE_ENFORCE_CUDA_SUCCESS(
                    platform::dynload::hipdnnConvolutionForward(
                        handle, &alpha, args2.idesc.desc(),
                        x + i * group_offset_in, args2.wdesc.desc(),
                        ddw + i * group_offset_filter, args2.cdesc.desc(),
                        fwd_algo2, workspace_ptr, workspace_size, &alpha,
                        args2.odesc.desc(),
                        transformed_ddy_channel + i * group_offset_out));
              },
              workspace_size);
        }
      }
      if (channel_last) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_ddO_channel, ddO);
      }
    }
    T* transformed_dy_channel = transformed_dO_channel.data<T>();
    if (dW && ddX) {
      ddx = transformed_ddX.data<T>();
      for (int i = 0; i < groups; i++) {
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::hipdnnConvolutionBackwardFilter(
                      handle, &alpha, args3.idesc.desc(),
                      ddx + i * group_offset_in, args3.odesc.desc(),
                      transformed_dy_channel + i * group_offset_out,
                      args3.cdesc.desc(), filter_algo, workspace_ptr,
                      workspace_size, &beta, args3.wdesc.desc(),
                      dw + i * group_offset_filter));
            },
            workspace_size);
      }
    }

    if (dX && ddW) {
      ddw = ddW->data<T>();
      for (int i = 0; i < groups; i++) {
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::hipdnnConvolutionBackwardData(
                      handle, &alpha, args4.wdesc.desc(),
                      ddw + i * group_offset_filter, args4.odesc.desc(),
                      transformed_dy_channel + i * group_offset_out,
                      args4.cdesc.desc(), data_algo, workspace_ptr,
                      workspace_size, &beta, args4.idesc.desc(),
                      transformed_dx + i * group_offset_in));
            },
            workspace_size);
      }

      if (!is_sys_pad) {
        // reverse padded input
        std::vector<int> starts(X->dims().size(), 0);
        std::vector<int> axes(X->dims().size(), 0);

        for (size_t i = 0; i < X->dims().size(); ++i) {
          starts[i] = input_pad[2 * i];
          axes[i] = i;
        }
        if (X->dims().size() == 4) {
          RemovePaddingSlice<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, &transformed_dX, &transformed_dX_channel, starts, axes);
        } else {
          RemovePaddingSlice<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, &transformed_dX, &transformed_dX_channel, starts, axes);
        }
      }
      if (channel_last) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_dX_channel, dX);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace plat = paddle::platform;
REGISTER_OP_KERNEL(conv2d, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvOpKernel<float>,
                   paddle::operators::CUDNNConvOpKernel<double>,
                   paddle::operators::CUDNNConvOpKernel<plat::float16>);
REGISTER_OP_KERNEL(conv2d_grad, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvGradOpKernel<float>,
                   paddle::operators::CUDNNConvGradOpKernel<double>,
                   paddle::operators::CUDNNConvGradOpKernel<plat::float16>);
REGISTER_OP_KERNEL(
    conv2d_grad_grad, CUDNN, plat::CUDAPlace,
    paddle::operators::CUDNNConvDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<double>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<plat::float16>);

REGISTER_OP_CUDA_KERNEL(
    depthwise_conv2d_grad_grad,
    paddle::operators::CUDNNConvDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<double>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<plat::float16>);

REGISTER_OP_KERNEL(conv3d, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvOpKernel<float>,
                   paddle::operators::CUDNNConvOpKernel<double>,
                   paddle::operators::CUDNNConvOpKernel<plat::float16>);
REGISTER_OP_KERNEL(conv3d_grad, CUDNN, plat::CUDAPlace,
                   paddle::operators::CUDNNConvGradOpKernel<float>,
                   paddle::operators::CUDNNConvGradOpKernel<double>);
REGISTER_OP_KERNEL(
    conv3d_grad_grad, CUDNN, plat::CUDAPlace,
    paddle::operators::CUDNNConvDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<double>,
    paddle::operators::CUDNNConvDoubleGradOpKernel<plat::float16>);
