#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// Part of the following code in this file refs to
// https://github.com/msracver/Deformable-ConvNets/blob/master/DCNv2_op/nn/modulated_deformable_im2col.cuh
//
// Copyright (c) 2018 Microsoft
// Licensed under The MIT License [see LICENSE for details]
// \file modulated_deformable_im2col.cuh
// \brief
// \author Yuwen Xiong, Haozhi Qi, Jifeng Dai, Xizhou Zhu, Han Hu

#include <algorithm>
#include <vector>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/deformable_conv_op.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__device__ T DmcnGetGradientWeight(T argmax_h, T argmax_w, const int h,
                                   const int w, const int height,
                                   const int width) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  T weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename T>
__global__ void ModulatedDeformableCol2imGpuKernel(
    const int nthreads, const T* data_col, const T* data_offset,
    const T* data_mask, const int channels, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int channel_per_deformable_group,
    const int batch_size, const int deformable_group, const int height_col,
    const int width_col, T* grad_im) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t thread = index; thread < nthreads; thread += offset) {
    const int j = (thread / width_col / height_col / batch_size) % kernel_w;
    const int i =
        (thread / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c =
        thread / width_col / height_col / batch_size / kernel_w / kernel_h;

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = thread % width_col;
    int h_out = (thread / width_col) % height_col;
    int b = (thread / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const T* data_offset_ptr = data_offset +
                               (b * deformable_group + deformable_group_index) *
                                   2 * kernel_h * kernel_w * height_col *
                                   width_col;
    const T* data_mask_ptr = data_mask +
                             (b * deformable_group + deformable_group_index) *
                                 kernel_h * kernel_w * height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const int data_mask_hw_ptr =
        ((i * kernel_w + j) * height_col + h_out) * width_col + w_out;
    const T offset_h = data_offset_ptr[data_offset_h_ptr];
    const T offset_w = data_offset_ptr[data_offset_w_ptr];
    const T mask = data_mask_ptr[data_mask_hw_ptr];
    const T cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const T cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const T cur_top_grad = data_col[thread] * mask;
    const int cur_h = static_cast<int>(cur_inv_h_data);
    const int cur_w = static_cast<int>(cur_inv_w_data);
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          T weight =
              DmcnGetGradientWeight(cur_inv_h_data, cur_inv_w_data, cur_h + dy,
                                    cur_w + dx, height, width);

          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

template <typename T>
inline void ModulatedDeformableCol2im(
    const platform::DeviceContext& ctx, const T* data_col, const T* data_offset,
    const T* data_mask, const std::vector<int64_t> im_shape,
    const std::vector<int64_t> col_shape,
    const std::vector<int64_t> kernel_shape, const std::vector<int> pad,
    const std::vector<int> stride, const std::vector<int> dilation,
    const int deformable_group, T* grad_im) {
  int channel_per_deformable_group = im_shape[0] / deformable_group;
  int num_kernels = col_shape[0] * col_shape[1] * col_shape[2] * col_shape[3];
  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  ModulatedDeformableCol2imGpuKernel<T><<<
      blocks, threads, 0,
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
      num_kernels, data_col, data_offset, data_mask, im_shape[0], im_shape[1],
      im_shape[2], kernel_shape[2], kernel_shape[3], pad[0], pad[1], stride[0],
      stride[1], dilation[0], dilation[1], channel_per_deformable_group,
      col_shape[1], deformable_group, col_shape[2], col_shape[3], grad_im);
}

template <typename T>
__device__ T DmcnGetCoordinateWeight(T argmax_h, T argmax_w, const int height,
                                     const int width, const T* im_data,
                                     const int data_width, const int bp_dir) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  T weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) *
                im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) *
                im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) *
                im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) *
                im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) *
                im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) *
                im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) *
                im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) *
                im_data[argmax_h_high * data_width + argmax_w_high];
  }
  return weight;
}

template <typename T>
__device__ T DmcnIm2colBilinear(const T* bottom_data, const int data_width,
                                const int height, const int width, T h, T w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  T lh = h - h_low;
  T lw = w - w_low;
  T hh = 1 - lh, hw = 1 - lw;

  T v1 = 0;
  if (h_low >= 0 && w_low >= 0) v1 = bottom_data[h_low * data_width + w_low];
  T v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  T v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  T v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  T w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename T>
__global__ void ModulatedDeformableCol2imCoordGpuKernel(
    const int nthreads, const T* data_col, const T* data_im,
    const T* data_offset, const T* data_mask, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int batch_size,
    const int offset_channels, const int deformable_group, const int height_col,
    const int width_col, T* grad_offset, T* grad_mask) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    T val = 0, mval = 0;
    const int w = i % width_col;
    const int h = (i / width_col) % height_col;
    const int c = (i / width_col / height_col) % offset_channels;
    const int b = (i / width_col / height_col) / offset_channels;

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const T* data_col_ptr = data_col +
                            deformable_group_index *
                                channel_per_deformable_group * batch_size *
                                width_col * height_col;
    const T* data_im_ptr = data_im +
                           (b * deformable_group + deformable_group_index) *
                               channel_per_deformable_group / kernel_h /
                               kernel_w * height * width;
    const T* data_offset_ptr = data_offset +
                               (b * deformable_group + deformable_group_index) *
                                   2 * kernel_h * kernel_w * height_col *
                                   width_col;
    const T* data_mask_ptr = data_mask +
                             (b * deformable_group + deformable_group_index) *
                                 kernel_h * kernel_w * height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = offset_c / 2; col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos =
          (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i =
          (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const int data_mask_hw_ptr =
          (((i * kernel_w + j) * height_col + h_out) * width_col + w_out);
      const T offset_h = data_offset_ptr[data_offset_h_ptr];
      const T offset_w = data_offset_ptr[data_offset_w_ptr];
      const T mask = data_mask_ptr[data_mask_hw_ptr];
      T inv_h = h_in + i * dilation_h + offset_h;
      T inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -2;
      } else {
        mval += data_col_ptr[col_pos] *
                DmcnIm2colBilinear(data_im_ptr + cnt * height * width, width,
                                   height, width, inv_h, inv_w);
      }
      const T weight = DmcnGetCoordinateWeight(
          inv_h, inv_w, height, width, data_im_ptr + cnt * height * width,
          width, bp_dir);
      val += weight * data_col_ptr[col_pos] * mask;
      cnt += 1;
    }
    grad_offset[i] = val;
    if (offset_c % 2 == 0)
      grad_mask[(((b * deformable_group + deformable_group_index) * kernel_h *
                      kernel_w +
                  offset_c / 2) *
                     height_col +
                 h) *
                    width_col +
                w] = mval;
  }
}

template <typename T>
inline void ModulatedDeformableCol2imCoord(
    const platform::DeviceContext& ctx, const T* data_col, const T* data_im,
    const T* data_offset, const T* data_mask,
    const std::vector<int64_t> im_shape, const std::vector<int64_t> col_shape,
    const std::vector<int64_t> kernel_shape, const std::vector<int> paddings,
    const std::vector<int> strides, const std::vector<int> dilations,
    const int deformable_groups, T* grad_offset, T* grad_mask) {
  int num_kernels = 2 * kernel_shape[2] * kernel_shape[3] * col_shape[1] *
                    col_shape[2] * col_shape[3] * deformable_groups;
  int channel_per_deformable_group = col_shape[0] / deformable_groups;
  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  ModulatedDeformableCol2imCoordGpuKernel<T><<<
      blocks, threads, 0,
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
      num_kernels, data_col, data_im, data_offset, data_mask, im_shape[0],
      im_shape[1], im_shape[2], kernel_shape[2], kernel_shape[3], paddings[0],
      paddings[1], strides[0], strides[1], dilations[0], dilations[1],
      channel_per_deformable_group, col_shape[1],
      2 * kernel_shape[2] * kernel_shape[3] * deformable_groups,
      deformable_groups, col_shape[2], col_shape[3], grad_offset, grad_mask);
}

template <typename T>
__global__ void ModulatedDeformableIm2colGpuKernel(
    const int nthreads, const T* data_im, const T* data_offset,
    const T* data_mask, const int height, const int width, const int kernel_h,
    const int kernel_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int batch_size,
    const int num_channels, const int deformable_group, const int height_col,
    const int width_col, T* data_col) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    const int w_col = i % width_col;
    const int h_col = (i / width_col) % height_col;
    const int b_col = (i / width_col) / height_col % batch_size;
    const int c_im = (i / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    T* data_col_ptr =
        data_col +
        ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const T* data_im_ptr =
        data_im + (b_col * num_channels + c_im) * height * width;
    const T* data_offset_ptr =
        data_offset +
        (b_col * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;
    const T* data_mask_ptr =
        data_mask +
        (b_col * deformable_group + deformable_group_index) * kernel_h *
            kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;

        const T offset_h = data_offset_ptr[data_offset_h_ptr];
        const T offset_w = data_offset_ptr[data_offset_w_ptr];
        const T mask = data_mask_ptr[data_mask_hw_ptr];
        T val = static_cast<T>(0);
        const T h_im = h_in + i * dilation_h + offset_h;
        const T w_im = w_in + j * dilation_w + offset_w;
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
          val =
              DmcnIm2colBilinear(data_im_ptr, width, height, width, h_im, w_im);
        }
        *data_col_ptr = val * mask;
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

template <typename T>
inline void ModulatedDeformableIm2col(
    const platform::DeviceContext& ctx, const T* data_im, const T* data_offset,
    const T* data_mask, const std::vector<int64_t> im_shape,
    const std::vector<int64_t> col_shape,
    const std::vector<int64_t> filter_shape, const std::vector<int> paddings,
    const std::vector<int> strides, const std::vector<int> dilations,
    const int deformable_groups, T* data_col) {
  int channel_per_deformable_group = im_shape[0] / deformable_groups;
  int num_kernels = im_shape[0] * col_shape[1] * col_shape[2] * col_shape[3];

  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  ModulatedDeformableIm2colGpuKernel<T><<<
      blocks, threads, 0,
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
      num_kernels, data_im, data_offset, data_mask, im_shape[1], im_shape[2],
      filter_shape[2], filter_shape[3], paddings[0], paddings[1], strides[0],
      strides[1], dilations[0], dilations[1], channel_per_deformable_group,
      col_shape[1], im_shape[0], deformable_groups, col_shape[2], col_shape[3],
      data_col);
}

template <typename T>
__global__ void FilterGradAddupGpuKernel(const int nthreads, const int n,
                                         const int height, const int width,
                                         const T* dweight_3d, T* filter_grad) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    filter_grad[i] = filter_grad[i] + dweight_3d[i];
  }
}

template <typename DeviceContext, typename T>
class DeformableConvCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const Tensor* input = ctx.Input<Tensor>("Input");
    const Tensor offset = *ctx.Input<Tensor>("Offset");
    const Tensor mask = *ctx.Input<Tensor>("Mask");
    Tensor filter = *ctx.Input<Tensor>("Filter");
    Tensor* output = ctx.Output<Tensor>("Output");
    output->mutable_data<T>(ctx.GetPlace());

    auto& dev_ctx = ctx.cuda_device_context();

    const int groups = ctx.Attr<int>("groups");
    const int deformable_groups = ctx.Attr<int>("deformable_groups");
    const int im2col_step = ctx.Attr<int>("im2col_step");
    const std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    const std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    const std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");

    const int batch_size = static_cast<int>(input->dims()[0]);

    std::vector<int64_t> filter_shape_vec(framework::vectorize(filter.dims()));
    std::vector<int64_t> output_shape_vec(framework::vectorize(output->dims()));

    // col_shape_vec: {c_i * k_h * k_w, im2col_step, o_h, o_w}
    std::vector<int64_t> col_buffer_shape_vec(filter_shape_vec.size());
    col_buffer_shape_vec[0] =
        input->dims()[1] * filter.dims()[2] * filter.dims()[3];
    col_buffer_shape_vec[1] = im2col_step;
    for (size_t j = 0; j < filter_shape_vec.size() - 2; ++j) {
      col_buffer_shape_vec[j + 2] = output_shape_vec[j + 2];
    }
    framework::DDim col_shape(framework::make_ddim(col_buffer_shape_vec));
    std::vector<int64_t> output_buffer_shape_vec(1);
    output_buffer_shape_vec[0] = batch_size * output_shape_vec[1] *
                                 output_shape_vec[2] * output_shape_vec[3];
    framework::DDim output_shape(framework::make_ddim(output_buffer_shape_vec));
    Tensor col_buffer;
    Tensor output_buffer;
    col_buffer = ctx.AllocateTmpTensor<T, DeviceContext>(col_shape, dev_ctx);
    output_buffer =
        ctx.AllocateTmpTensor<T, DeviceContext>(output_shape, dev_ctx);

    int64_t M = output_shape_vec[1] / groups;
    int64_t N = im2col_step * output_shape_vec[2] * output_shape_vec[3];
    int64_t K =
        input->dims()[1] * filter_shape_vec[2] * filter_shape_vec[3] / groups;

    Tensor weight_3d;
    weight_3d.ShareDataWith(filter).Resize(
        framework::make_ddim({groups, M, K}));
    Tensor col_buffer_3d;
    col_buffer_3d.ShareDataWith(col_buffer)
        .Resize(framework::make_ddim({groups, K, N}));
    Tensor output_4d;
    output_4d.ShareDataWith(output_buffer)
        .Resize(framework::make_ddim({batch_size / im2col_step, groups, M, N}));
    output_4d.mutable_data<T>(ctx.GetPlace());
    framework::DDim input_shape =
        framework::slice_ddim(input->dims(), 1, input->dims().size());
    std::vector<int64_t> input_shape_vec = framework::vectorize(input_shape);

    int input_dim = input->numel() / input->dims()[0];
    int input_offset_dim = offset.numel() / offset.dims()[0];
    int input_mask_dim = mask.numel() / mask.dims()[0];

    auto blas = math::GetBlas<DeviceContext, T>(dev_ctx);

    const T* input_ptr = input->data<T>();
    const T* offset_ptr = offset.data<T>();
    const T* mask_ptr = mask.data<T>();
    col_buffer.mutable_data<T>(ctx.GetPlace());
    T* col_buffer_ptr = col_buffer.data<T>();

    for (int i = 0; i < batch_size / im2col_step; ++i) {
      ModulatedDeformableIm2col(
          ctx.device_context(), input_ptr + i * im2col_step * input_dim,
          offset_ptr + i * im2col_step * input_offset_dim,
          mask_ptr + i * im2col_step * input_mask_dim, input_shape_vec,
          col_buffer_shape_vec, filter_shape_vec, paddings, strides, dilations,
          deformable_groups, col_buffer_ptr);

      Tensor output_3d = output_4d.Slice(i, i + 1).Resize(
          framework::slice_ddim(output_4d.dims(), 1, output_4d.dims().size()));
      for (int g = 0; g < groups; ++g) {
        Tensor weight_3d_slice =
            weight_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                weight_3d.dims(), 1, weight_3d.dims().size()));
        Tensor col_buffer_3d_slice =
            col_buffer_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                col_buffer_3d.dims(), 1, col_buffer_3d.dims().size()));
        Tensor output_3d_slice =
            output_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                output_3d.dims(), 1, output_3d.dims().size()));

        blas.MatMul(weight_3d_slice, false, col_buffer_3d_slice, false, T(1.0),
                    &output_3d_slice, T(0.0));
      }
    }
    output->ShareDataWith(output_buffer)
        .Resize(framework::make_ddim(output_shape_vec));
  }
};

template <typename DeviceContext, typename T>
class DeformableConvGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const Tensor* output_grad =
        ctx.Input<Tensor>(framework::GradVarName("Output"));
    Tensor* input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    Tensor* filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));
    Tensor* offset_grad = ctx.Output<Tensor>(framework::GradVarName("Offset"));
    Tensor* mask_grad = ctx.Output<Tensor>(framework::GradVarName("Mask"));

    const Tensor* input = ctx.Input<Tensor>("Input");
    Tensor offset = *ctx.Input<Tensor>("Offset");
    Tensor mask = *ctx.Input<Tensor>("Mask");
    Tensor filter = *ctx.Input<Tensor>("Filter");
    if (!input_grad && !filter_grad && !offset_grad && !mask_grad) return;

    int groups = ctx.Attr<int>("groups");
    int deformable_groups = ctx.Attr<int>("deformable_groups");
    int im2col_step = ctx.Attr<int>("im2col_step");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");

    auto& dev_ctx = ctx.cuda_device_context();
    const int batch_size = static_cast<int>(input->dims()[0]);

    framework::DDim input_shape =
        framework::slice_ddim(input->dims(), 1, input->dims().size());
    std::vector<int64_t> input_shape_vec = framework::vectorize(input_shape);
    std::vector<int64_t> filter_shape_vec(framework::vectorize(filter.dims()));
    std::vector<int64_t> output_shape_vec(
        framework::vectorize(output_grad->dims()));

    std::vector<int64_t> col_buffer_shape_vec(filter_shape_vec.size());
    col_buffer_shape_vec[0] =
        input->dims()[1] * filter.dims()[2] * filter.dims()[3];
    col_buffer_shape_vec[1] = im2col_step;
    for (size_t j = 0; j < filter_shape_vec.size() - 2; ++j) {
      col_buffer_shape_vec[j + 2] = output_shape_vec[j + 2];
    }
    framework::DDim col_shape(framework::make_ddim(col_buffer_shape_vec));
    std::vector<int64_t> output_buffer_shape_vec(1);
    output_buffer_shape_vec[0] = batch_size * output_shape_vec[1] *
                                 output_shape_vec[2] * output_shape_vec[3];
    framework::DDim output_shape(framework::make_ddim(output_buffer_shape_vec));
    Tensor col_buffer;
    Tensor output_buffer;
    col_buffer = ctx.AllocateTmpTensor<T, DeviceContext>(col_shape, dev_ctx);
    output_buffer =
        ctx.AllocateTmpTensor<T, DeviceContext>(output_shape, dev_ctx);

    output_buffer.ShareDataWith(*output_grad);

    int64_t M =
        input_shape_vec[0] / groups * filter_shape_vec[2] * filter_shape_vec[3];
    int64_t N = im2col_step * output_shape_vec[2] * output_shape_vec[3];
    int64_t K = output_shape_vec[1] / groups;

    framework::DDim weight_3d_shape = {groups, K, M};
    framework::DDim out_grad_4d_shape = {batch_size / im2col_step, groups, K,
                                         N};
    framework::DDim col_buffer_3d_shape = {groups, M, N};
    framework::DDim filter_grad_shape = {groups, K, M};

    Tensor weight_3d;
    weight_3d.ShareDataWith(filter).Resize(weight_3d_shape);
    Tensor out_grad_4d;
    out_grad_4d.ShareDataWith(output_buffer).Resize(out_grad_4d_shape);
    Tensor col_buffer_3d;
    col_buffer_3d.ShareDataWith(col_buffer).Resize(col_buffer_3d_shape);

    math::SetConstant<DeviceContext, T> set_zero;
    auto blas = math::GetBlas<DeviceContext, T>(dev_ctx);

    col_buffer.mutable_data<T>(ctx.GetPlace());
    col_buffer_3d.mutable_data<T>(ctx.GetPlace());
    out_grad_4d.mutable_data<T>(ctx.GetPlace());

    int input_dim = input->numel() / input->dims()[0];
    int input_offset_dim = offset.numel() / offset.dims()[0];
    int input_mask_dim = mask.numel() / mask.dims()[0];

    if (filter_grad) {
      filter_grad->mutable_data<T>(ctx.GetPlace());
      filter_grad->Resize(filter_grad_shape);
      set_zero(dev_ctx, filter_grad, static_cast<T>(0));
    }

    if (input_grad) {
      input_grad->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, input_grad, static_cast<T>(0));
    }

    if (offset_grad && mask_grad) {
      offset_grad->mutable_data<T>(ctx.GetPlace());
      mask_grad->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, offset_grad, static_cast<T>(0));
      set_zero(dev_ctx, mask_grad, static_cast<T>(0));
    }

    for (int i = 0; i < batch_size / im2col_step; ++i) {
      Tensor out_grad_3d =
          out_grad_4d.Slice(i, i + 1).Resize(framework::slice_ddim(
              out_grad_4d.dims(), 1, out_grad_4d.dims().size()));
      for (int g = 0; g < groups; ++g) {
        Tensor weight_3d_slice =
            weight_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                weight_3d.dims(), 1, weight_3d.dims().size()));
        Tensor out_grad_3d_slice =
            out_grad_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                out_grad_3d.dims(), 1, out_grad_3d.dims().size()));
        Tensor col_buffer_3d_slice =
            col_buffer_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                col_buffer_3d.dims(), 1, col_buffer_3d.dims().size()));

        blas.MatMul(weight_3d_slice, true, out_grad_3d_slice, false, T(1.0),
                    &col_buffer_3d_slice, T(0.0));
      }
      col_buffer.Resize(col_shape);

      T* col_buffer_ptr = col_buffer.data<T>();
      const T* input_ptr = input->data<T>();
      const T* offset_ptr = offset.data<T>();
      const T* mask_ptr = mask.data<T>();

      if (mask_grad && offset_grad) {
        T* offset_grad_ptr = offset_grad->data<T>();
        T* mask_grad_ptr = mask_grad->data<T>();
        ModulatedDeformableCol2imCoord(
            ctx.device_context(), col_buffer_ptr,
            input_ptr + i * im2col_step * input_dim,
            offset_ptr + i * im2col_step * input_offset_dim,
            mask_ptr + i * im2col_step * input_mask_dim, input_shape_vec,
            col_buffer_shape_vec, filter_shape_vec, paddings, strides,
            dilations, deformable_groups,
            offset_grad_ptr + i * im2col_step * input_offset_dim,
            mask_grad_ptr + i * im2col_step * input_mask_dim);
      }
      if (input_grad) {
        T* input_grad_ptr = input_grad->data<T>();
        ModulatedDeformableCol2im(
            ctx.device_context(), col_buffer_ptr,
            offset_ptr + i * im2col_step * input_offset_dim,
            mask_ptr + i * im2col_step * input_mask_dim, input_shape_vec,
            col_buffer_shape_vec, filter_shape_vec, paddings, strides,
            dilations, deformable_groups,
            input_grad_ptr + i * im2col_step * input_dim);
        input_grad->Resize(input->dims());
      }

      ModulatedDeformableIm2col(
          ctx.device_context(), input_ptr + i * im2col_step * input_dim,
          offset_ptr + i * im2col_step * input_offset_dim,
          mask_ptr + i * im2col_step * input_mask_dim, input_shape_vec,
          col_buffer_shape_vec, filter_shape_vec, paddings, strides, dilations,
          deformable_groups, col_buffer_ptr);

      col_buffer_3d.Resize(col_buffer_3d_shape);

      if (filter_grad) {
        Tensor dweight_3d;
        dweight_3d =
            ctx.AllocateTmpTensor<T, DeviceContext>(filter_grad_shape, dev_ctx);
        for (int g = 0; g < groups; ++g) {
          Tensor out_grad_3d_slice =
              out_grad_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                  out_grad_3d.dims(), 1, out_grad_3d.dims().size()));
          Tensor col_buffer_3d_slice =
              col_buffer_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                  col_buffer_3d.dims(), 1, col_buffer_3d.dims().size()));
          Tensor dweight_3d_slice =
              dweight_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                  dweight_3d.dims(), 1, dweight_3d.dims().size()));

          blas.MatMul(out_grad_3d_slice, false, col_buffer_3d_slice, true,
                      T(1.0), &dweight_3d_slice, T(0.0));
        }
        FilterGradAddupGpuKernel<
            T><<<NumBlocks(dweight_3d.numel()), kNumCUDAThreads, 0,
                 ctx.cuda_device_context().stream()>>>(
            dweight_3d.numel(), groups, K, M, dweight_3d.data<T>(),
            filter_grad->data<T>());
      }
    }
    if (filter_grad) {
      filter_grad->Resize(filter.dims());
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;

REGISTER_OP_CUDA_KERNEL(deformable_conv,
                        ops::DeformableConvCUDAKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(deformable_conv_grad,
                        ops::DeformableConvGradCUDAKernel<CUDA, float>);
