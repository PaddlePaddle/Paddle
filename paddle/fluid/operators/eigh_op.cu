#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifndef PADDLE_WITH_HIP
// HIP not support cusolver

#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/eigh_op.h"
#include "paddle/fluid/platform/dynload/cusolver.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename ValueType, typename T>
struct MatrixEighFunctor {
 public:
  void operator()(const framework::ExecutionContext &ctx, const Tensor &input,
                  Tensor *eigen_values, Tensor *eigen_vectors, bool is_lower,
                  bool compute_v) {
    auto *out_value = eigen_values->mutable_data<ValueType>(ctx.GetPlace());
    auto *out_vector = eigen_vectors->mutable_data<T>(ctx.GetPlace());

    auto &dims = input.dims();
    int dim_size = dims.size();
    int64_t batch_size = GetBatchSize(dims);

    hipblasFillMode_t uplo =
        is_lower ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
    hipsolverEigMode_t jobz =
        compute_v ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;

    int n = dims[dim_size - 1];
    int lda = std::max<int>(1, n);
    auto vector_stride = dims[dim_size - 1] * dims[dim_size - 2];
    auto values_stride = dims[dim_size - 1];

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto dito = DeviceIndependenceTensorOperations<platform::CUDADeviceContext,
                                                   T, ValueType>(ctx);
    Tensor output_v_var_trans = dito.Transpose(input);
    TensorCopy(output_v_var_trans, ctx.GetPlace(), eigen_vectors);

    int lwork = 0;
    auto info = memory::Alloc(dev_ctx, sizeof(int) * batch_size);
    auto *info_ptr = reinterpret_cast<int *>(info->ptr());

    bool flag = (eigen_vectors->type() == framework::proto::VarType::FP32 &&
                 values_stride >= 32 && values_stride <= 512);

    hipsolverSyevjInfo_t syevj_params;
    if (flag) {
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipsolverDnCreateSyevjInfo(&syevj_params));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipsolverDnSsyevj_bufferSize(
              dev_ctx.cusolver_dn_handle(), jobz, uplo, n,
              reinterpret_cast<const float *>(out_vector), lda,
              reinterpret_cast<const float *>(out_value), &lwork,
              syevj_params));
    } else {
      EvdBuffer(dev_ctx.cusolver_dn_handle(), jobz, uplo, n, out_vector, lda,
                out_value, &lwork);
    }
    auto work = memory::Alloc(dev_ctx, sizeof(T) * lwork);
    auto *work_ptr = reinterpret_cast<T *>(work->ptr());

    for (auto i = 0; i < batch_size; i++) {
      auto vector_data = out_vector + i * vector_stride;
      auto value_data = out_value + i * values_stride;
      auto handle = dev_ctx.cusolver_dn_handle();
      if (flag) {
        PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipsolverDnSsyevj(
            handle, jobz, uplo, n, reinterpret_cast<float *>(vector_data), lda,
            reinterpret_cast<float *>(value_data),
            reinterpret_cast<float *>(work_ptr), lwork, info_ptr,
            syevj_params));
      } else {
        Evd(handle, jobz, uplo, n, vector_data, lda, value_data, work_ptr,
            lwork, info_ptr);
      }

      int error_info;
      memory::Copy(platform::CPUPlace(), &error_info,
                   BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                   info_ptr, sizeof(int), dev_ctx.stream());
      PADDLE_ENFORCE_EQ(
          error_info, 0,
          platform::errors::PreconditionNotMet(
              "For batch [%d]: the [%d] argument had an illegal value", i,
              error_info));
    }

    if (flag) {
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipsolverDnDestroySyevjInfo(syevj_params));
    }

    if (compute_v) {
      *eigen_vectors = dito.Transpose(*eigen_vectors);
    }
  }

  void EvdBuffer(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
                 hipblasFillMode_t uplo, int n, const T *A, int lda,
                 const ValueType *W, int *lwork) const;

  void Evd(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
           hipblasFillMode_t uplo, int n, T *A, int lda, ValueType *W, T *work,
           int lwork, int *devInfo) const;
};

template <typename ValueType, typename T>
class EighGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto input_var = ctx.Input<Tensor>("X");
    auto output_w_var = ctx.Output<Tensor>("Eigenvalues");
    auto output_v_var = ctx.Output<Tensor>("Eigenvectors");
    std::string lower = ctx.Attr<std::string>("UPLO");
    bool is_lower = (lower == "L");
    MatrixEighFunctor<ValueType, T> functor;
    functor(ctx, *input_var, output_w_var, output_v_var, is_lower, true);
  }
};

#define FUNC_WITH_TYPES(m)                                       \
  m(float, float, Ssy, float) m(double, double, Dsy, double)     \
      m(float, paddle::platform::complex<float>, Che, hipComplex) \
          m(double, paddle::platform::complex<double>, Zhe, hipDoubleComplex)

#define EVDBUFFER_INSTANCE(ValueType, T, C, CastType)                          \
  template <>                                                                  \
  void MatrixEighFunctor<ValueType, T>::EvdBuffer(                             \
      hipsolverHandle_t handle, hipsolverEigMode_t jobz,                       \
      hipblasFillMode_t uplo, int n, const T *A, int lda, const ValueType *W,   \
      int *lwork) const {                                                      \
    PADDLE_ENFORCE_CUDA_SUCCESS(                                               \
        platform::dynload::cusolverDn##C##evd_bufferSize(                      \
            handle, jobz, uplo, n, reinterpret_cast<const CastType *>(A), lda, \
            W, lwork));                                                        \
  }

FUNC_WITH_TYPES(EVDBUFFER_INSTANCE);

#define EVD_INSTANCE(ValueType, T, C, CastType)                           \
  template <>                                                             \
  void MatrixEighFunctor<ValueType, T>::Evd(                              \
      hipsolverHandle_t handle, hipsolverEigMode_t jobz,                  \
      hipblasFillMode_t uplo, int n, T *A, int lda, ValueType *W, T *work, \
      int lwork, int *devInfo) const {                                    \
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::cusolverDn##C##evd(    \
        handle, jobz, uplo, n, reinterpret_cast<CastType *>(A), lda, W,   \
        reinterpret_cast<CastType *>(work), lwork, devInfo));             \
  }

FUNC_WITH_TYPES(EVD_INSTANCE);

#undef FUNC_WITH_TYPES
#undef EVDBUFFER_INSTANCE
#undef EVD_INSTANCE

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    eigh, ops::EighGPUKernel<float, float>, ops::EighGPUKernel<double, double>,
    ops::EighGPUKernel<float, paddle::platform::complex<float>>,
    ops::EighGPUKernel<double, paddle::platform::complex<double>>);

REGISTER_OP_CUDA_KERNEL(
    eigh_grad,
    ops::EighGradKernel<paddle::platform::CUDADeviceContext, float, float>,
    ops::EighGradKernel<paddle::platform::CUDADeviceContext, double, double>,
    ops::EighGradKernel<paddle::platform::CUDADeviceContext, float,
                        paddle::platform::complex<float>>,
    ops::EighGradKernel<paddle::platform::CUDADeviceContext, double,
                        paddle::platform::complex<double>>);

#endif  // not PADDLE_WITH_HIP
