#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/eigh_op.h"
#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/dynload/cusolver.h"

namespace paddle {
namespace operators {

template <typename T, typename ValueType>
void getBufferSize(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
                   hipblasFillMode_t uplo, int n, const T *A, int lda,
                   const ValueType *W, int *lwork);

template <>
void getBufferSize<float, float>(hipsolverHandle_t handle,
                                 hipsolverEigMode_t jobz, hipblasFillMode_t uplo,
                                 int n, const float *A, int lda, const float *W,
                                 int *lwork) {
  hipsolverDnSsyevd_bufferSize(handle, jobz, uplo, n, A, lda, W, lwork);
}

template <>
void getBufferSize<double, double>(hipsolverHandle_t handle,
                                   hipsolverEigMode_t jobz,
                                   hipblasFillMode_t uplo, int n,
                                   const double *A, int lda, const double *W,
                                   int *lwork) {
  hipsolverDnDsyevd_bufferSize(handle, jobz, uplo, n, A, lda, W, lwork);
}

template <>
void getBufferSize<paddle::platform::complex<float>, float>(
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo,
    int n, const paddle::platform::complex<float> *A, int lda, const float *W,
    int *lwork) {
  hipsolverDnCheevd_bufferSize(handle, jobz, uplo, n,
                              reinterpret_cast<const hipComplex *>(A), lda, W,
                              lwork);
}

template <>
void getBufferSize<paddle::platform::complex<double>, double>(
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo,
    int n, const paddle::platform::complex<double> *A, int lda, const double *W,
    int *lwork) {
  hipsolverDnZheevd_bufferSize(handle, jobz, uplo, n,
                              reinterpret_cast<const hipDoubleComplex *>(A), lda,
                              W, lwork);
}

template <typename T, typename ValueType>
void computeValues(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
                   hipblasFillMode_t uplo, int n, T *A, int lda, ValueType *W,
                   T *work, int lwork, int *devInfo);

template <>
void computeValues<float, float>(hipsolverHandle_t handle,
                                 hipsolverEigMode_t jobz, hipblasFillMode_t uplo,
                                 int n, float *A, int lda, float *W,
                                 float *work, int lwork, int *devInfo) {
  hipsolverDnSsyevd(handle, jobz, uplo, n, A, lda, W, work, lwork, devInfo);
}

template <>
void computeValues<double, double>(hipsolverHandle_t handle,
                                   hipsolverEigMode_t jobz,
                                   hipblasFillMode_t uplo, int n, double *A,
                                   int lda, double *W, double *work, int lwork,
                                   int *devInfo) {
  hipsolverDnDsyevd(handle, jobz, uplo, n, A, lda, W, work, lwork, devInfo);
}

template <>
void computeValues<paddle::platform::complex<float>, float>(
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo,
    int n, paddle::platform::complex<float> *A, int lda, float *W,
    paddle::platform::complex<float> *work, int lwork, int *devInfo) {
  hipsolverDnCheevd(handle, jobz, uplo, n, reinterpret_cast<hipComplex *>(A), lda,
                   W, reinterpret_cast<hipComplex *>(work), lwork, devInfo);
}

template <>
void computeValues<paddle::platform::complex<double>, double>(
    hipsolverHandle_t handle, hipsolverEigMode_t jobz, hipblasFillMode_t uplo,
    int n, paddle::platform::complex<double> *A, int lda, double *W,
    paddle::platform::complex<double> *work, int lwork, int *devInfo) {
  hipsolverDnZheevd(handle, jobz, uplo, n,
                   reinterpret_cast<hipDoubleComplex *>(A), lda, W,
                   reinterpret_cast<hipDoubleComplex *>(work), lwork, devInfo);
}

using Tensor = framework::Tensor;

template <typename DeviceContext, typename T, typename ValueType>
class EighGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    // std::cout << "##########" << std::endl;
    const auto *input_var = ctx.Input<Tensor>("X");

    auto *output_w_var = ctx.Output<Tensor>("OutVector");
    auto *output_v_var = ctx.Output<Tensor>("OutValue");

    std::string lower = ctx.Attr<std::string>("UPLO");
    auto &dims = input_var->dims();
    int dim_size = dims.size();
    int64_t batch_size = 1;
    for (int i = 0; i < dims.size() - 2; i++) {
      batch_size *= dims[i];
    }
    auto *out_vector = output_w_var->mutable_data<T>(ctx.GetPlace());
    auto *out_value = output_v_var->mutable_data<ValueType>(ctx.GetPlace());

    hipblasFillMode_t uplo =
        (lower == "L") ? HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;

    int n = dims[dim_size - 1];
    int lda = std::max<int>(1, n);
    auto vector_stride = dims[dim_size - 1] * dims[dim_size - 2];
    auto values_stride = dims[dim_size - 1];
    paddle::framework::TensorCopy(
        *input_var, input_var->place(), dev_ctx,
        output_w_var);  // copy input data to temp data
    std::vector<int> axis(dim_size - 2);
    std::iota(axis.begin(), axis.end(), 0);
    axis.insert(axis.end(), {dim_size - 1, dim_size - 2});
    Tensor output_w_var_trans;
    output_w_var_trans.mutable_data<T>(dims, ctx.GetPlace());
    TransCompute<platform::CUDADeviceContext, T>(
        dim_size, dev_ctx, *output_w_var, &output_w_var_trans, axis);
    paddle::framework::TensorCopy(
        output_w_var_trans, output_w_var_trans.place(), dev_ctx, output_w_var);

    int lwork = 0;
    T *d_work = NULL;

    int *info_ptr = NULL;
    hipMalloc(reinterpret_cast<void **>(&info_ptr), sizeof(int));

    getBufferSize<T, ValueType>(dev_ctx.cusolver_dn_handle(), jobz, uplo, n,
                                out_vector, lda, out_value, &lwork);

    hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(T) * lwork);
    for (auto i = 0; i < batch_size; i++) {
      auto vector_data = out_vector + i * vector_stride;
      auto value_data = out_value + i * values_stride;
      // check the info
      // std::vector<T> error_info;
      // error_info.resize(4);
      // memory::Copy(platform::CPUPlace(), error_info.data(),
      //              BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
      //              out_vector, sizeof(T) * 4, dev_ctx.stream());
      // std::cout << error_info[0] << "\t" << error_info[1] << "\t" <<
      // error_info[2] << error_info[3] << "\n";
      auto handle = dev_ctx.cusolver_dn_handle();
      computeValues<T, ValueType>(handle, jobz, uplo, n, vector_data, lda,
                                  value_data, d_work, lwork, info_ptr);
    }

    // memory::Copy(platform::CPUPlace(), error_info.data(),
    //              BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
    //              info_ptr, sizeof(T) * batch_size, dev_ctx.stream());

    // for (int i = 0; i < batch_size; ++i) {
    //   PADDLE_ENFORCE_GT(error_info[i], 0,
    //               platform::errors::InvalidArgument(
    //                   "the [%d] argument had an illegal value",
    //                   error_info[i]));
    //   PADDLE_ENFORCE_LT(error_info[i], 0,
    //               platform::errors::InvalidArgument("if JOBZ = \'N\', [%d]
    //               off-diagonal elements of an intermediate tridiagonal form
    //               did not converge to zero; if JOBZ = \'V\', then the
    //               algorithm failed to compute an eigenvalue",
    //       error_info[i]));
    // }
    TransCompute<DeviceContext, T>(dim_size, dev_ctx, *output_w_var,
                                   &output_w_var_trans, axis);
    paddle::framework::TensorCopy(
        output_w_var_trans, output_w_var_trans.place(), dev_ctx, output_w_var);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    eigh, ops::EighGPUKernel<paddle::platform::CUDADeviceContext,
                             paddle::platform::complex<double>, double>,
    ops::EighGPUKernel<paddle::platform::CUDADeviceContext,
                       paddle::platform::complex<float>, float>,
    ops::EighGPUKernel<paddle::platform::CUDADeviceContext, double, double>,
    ops::EighGPUKernel<paddle::platform::CUDADeviceContext, float, float>);
