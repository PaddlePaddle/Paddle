#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/cvm_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void CvmComputeKernel(const bool use_cvm, const int64_t item_width,
                                 const T* X, T* Y, int64_t numel) {
  CUDA_KERNEL_LOOP(i, numel) {
    if (use_cvm) {
      if (i % item_width == 0) {
        Y[i] = log(X[i] + 1);
      } else if (i % item_width == 1) {
        Y[i] = log(X[i] + 1) - log(X[i - 1] + 1);
      } else {
        Y[i] = X[i];
      }
    } else {
      Y[i] = X[i / (item_width - 2) * item_width + i % (item_width - 2) + 2];
    }
  }
}

template <typename T>
__global__ void CvmGradComputeKernel(const bool use_cvm,
                                     const int64_t item_width, const T* CVM,
                                     const T* DY, T* DX, bool has_lod,
                                     const size_t* lod, size_t lod_size,
                                     int64_t numel) {
  CUDA_KERNEL_LOOP(i, numel) {
    int offset = i % item_width;
    if (offset <= 1) {
      int cvm_id = i / item_width;
      if (has_lod) {
        int low = 1;
        int high = lod_size - 1;
        while (low < high) {
          int mid = (low + high) / 2;
          if (cvm_id < lod[mid])
            high = mid;
          else
            low = mid + 1;
        }
        cvm_id = low - 1;
      }
      DX[i] = CVM[2 * cvm_id + offset];
    } else {
      if (use_cvm) {
        DX[i] = DY[i];
      } else {
        DX[i] = DY[i / item_width * (item_width - 2) + i % item_width - 2];
      }
    }
  }
}

template <typename T>
class CVMCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    const auto* x = context.Input<LoDTensor>("X");
    const T* x_data = x->data<T>();

    auto batch_size = x->dims()[0];
    auto numel = x->numel();
    auto item_size = numel / batch_size;
    auto use_cvm = context.Attr<bool>("use_cvm");

    auto* y = context.Output<LoDTensor>("Y");
    T* y_data = y->mutable_data<T>(context.GetPlace());

    // for Input X do not have Lod Information.
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    if (x->NumLevels() == 0) {
      CvmComputeKernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                             PADDLE_CUDA_NUM_THREADS,
                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          use_cvm, item_size, x_data, y_data, y->numel());
    } else {
      auto lod = x->lod()[0];
      PADDLE_ENFORCE_EQ(
          batch_size, lod[lod.size() - 1],
          platform::errors::PreconditionNotMet(
              "Input(X)'s dim[0] must be equal to last element of lod"));
      CvmComputeKernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                             PADDLE_CUDA_NUM_THREADS,
                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          use_cvm, item_size, x_data, y_data, y->numel());
    }
  }
};

template <typename T>
class CVMGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* dx = context.Output<LoDTensor>(framework::GradVarName("X"));
    T* dx_data = dx->mutable_data<T>(context.GetPlace());

    const Tensor* cvm = context.Input<Tensor>("CVM");
    const T* cvm_data = cvm->data<T>();

    const auto* dOut =
        context.Input<framework::LoDTensor>(framework::GradVarName("Y"));
    const T* dout_data = dOut->data<T>();

    auto use_cvm = context.Attr<bool>("use_cvm");

    auto offset = 2;
    auto batch_size = dx->dims()[0];
    auto dx_numel = dx->numel();
    auto item_size = dx_numel / batch_size;

    // for Input X do not have Lod Information.
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    if (dx->NumLevels() == 0) {
      CvmGradComputeKernel<<<(dx_numel + PADDLE_CUDA_NUM_THREADS - 1) /
                                 PADDLE_CUDA_NUM_THREADS,
                             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          use_cvm, item_size, cvm_data, dout_data, dx_data, false, NULL, 0,
          dx_numel);
    } else {
      auto lod = dx->lod()[0];
      PADDLE_ENFORCE_EQ(
          batch_size, lod[lod.size() - 1],
          platform::errors::PreconditionNotMet(
              "Output(X@GRAD)'s dim[0] must be equal to last element of lod"));
      CvmGradComputeKernel<<<(dx_numel + PADDLE_CUDA_NUM_THREADS - 1) /
                                 PADDLE_CUDA_NUM_THREADS,
                             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          use_cvm, item_size, cvm_data, dout_data, dx_data, true,
          lod.CUDAData(context.GetPlace()), lod.size(), dx_numel);
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(cvm, ops::CVMCUDAKernel<float>,
                        ops::CVMCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(cvm_grad, ops::CVMGradCUDAKernel<float>,
                        ops::CVMGradCUDAKernel<double>);
