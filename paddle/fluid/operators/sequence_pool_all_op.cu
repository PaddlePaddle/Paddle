#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/sequence_pool_all_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using LoDTensor = framework::LoDTensor;
using Tensor = framework::Tensor;

template <typename T, int BlockDim>
__global__ void sequence_pool_all_kernel(T **input, const T pad_value,
                                         size_t **all_lod, const int batch_size,
                                         const size_t item_dim, T **output) {
  int tid = blockIdx.x;  // tensor index
  int bid = blockIdx.y;  // batch index
  if (bid >= batch_size) return;
  size_t start = all_lod[tid][bid];
  size_t end = all_lod[tid][bid + 1];

  int offset = end - start;
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ou_storage;

  for (int i = blockIdx.z; i < item_dim; i += gridDim.z) {
    if (offset == 0) {
      output[tid][bid * item_dim + i] = pad_value;
    } else {
      T ou = static_cast<T>(0);
      for (int j = threadIdx.x; j < offset; j += blockDim.x) {
        const int index = j * item_dim + i;
        ou += static_cast<T>(input[tid][item_dim * start + index]);
      }
      ou = BlockReduce(ou_storage).Reduce(ou, hipcub::Sum());
      __syncthreads();
      if (threadIdx.x == 0) {
        output[tid][bid * item_dim + i] = ou;
      }
    }
  }
}

template <typename T>
struct SumPoolGradFunctor {
  HOSTDEVICE void operator()(const T *out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T *in_grad, const int *index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        in_grad[item_dim * i + tid] = out_grad[tid];
      }
    }
  }
};

template <typename T, typename Range_OP>
__global__ void sequence_pool_all_grad_kernel(Range_OP op, T **input,
                                              size_t **all_lod,
                                              const int batch_size,
                                              const size_t item_dim,
                                              T **output) {
  int tid = blockIdx.x;  // tensor index
  int bid = blockIdx.y;  // batch index
  if (bid >= batch_size) return;
  size_t start = all_lod[tid][bid];
  size_t end = all_lod[tid][bid + 1];

  op(&input[tid][bid * item_dim], start, end, item_dim, output[tid], nullptr);
}

template <typename DeviceContext, typename T>
class SequencePoolAllCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto in_vars = ctx.MultiInput<LoDTensor>("X");
    auto out_vars = ctx.MultiOutput<Tensor>("Out");
    std::string pooltype = ctx.Attr<std::string>("pooltype");
    PADDLE_ENFORCE_EQ(
        pooltype, "SUM",
        platform::errors::InvalidArgument(
            "Currently, it only supports SUM for sequence_pool_all op"));
    T pad_value = static_cast<T>(ctx.Attr<float>("pad_value"));

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    int batch_size = in_vars[0]->lod()[0].size() - 1;

    auto in_num = in_vars.size();
    std::vector<const size_t *> lod_data;
    for (int i = 0; i < in_num; ++i) {
      auto dims = in_vars[i]->dims();
      auto &lod = in_vars[i]->lod();
      PADDLE_ENFORCE_EQ(lod.size(), 1,
                        platform::errors::InvalidArgument(
                            "Currently, it only supports lod_level = 1."));
      PADDLE_ENFORCE_EQ(
          batch_size,
          /*batch size = */ static_cast<int64_t>(lod[0].size() - 1),
          platform::errors::InvalidArgument(
              "There are error of lod mesage in inputs."));
      PADDLE_ENFORCE_GE(
          dims[0],
          /*batch size = */ static_cast<int64_t>(lod[0].size() - 1),
          platform::errors::InvalidArgument("The first dimension of Input(X) "
                                            "must be large than batch size."));

      lod_data.emplace_back(lod[0].CUDAData(dev_ctx.GetPlace()));
      dims[0] = batch_size;
      out_vars[i]->Resize({dims});
      out_vars[i]->mutable_data<T>(ctx.GetPlace());
    }
    const size_t item_dim = out_vars[0]->numel() / out_vars[0]->dims()[0];

    std::vector<const T *> in_data;
    std::vector<T *> out_data;
    for (int i = 0; i < in_num; ++i) {
      in_data.emplace_back(in_vars[i]->data<T>());
      out_data.emplace_back(out_vars[i]->data<T>());
    }
    auto tmp_in_array = memory::Alloc(dev_ctx, in_data.size() * sizeof(T *));
    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_in_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(in_data.data()),
                 in_data.size() * sizeof(T *), dev_ctx.stream());
    T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());

    auto tmp_lod_array =
        memory::Alloc(dev_ctx, lod_data.size() * sizeof(size_t *));
    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_lod_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(lod_data.data()),
                 lod_data.size() * sizeof(size_t *), dev_ctx.stream());
    size_t **lod_array_data = reinterpret_cast<size_t **>(tmp_lod_array->ptr());

    auto tmp_out_array = memory::Alloc(dev_ctx, out_data.size() * sizeof(T *));
    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_out_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(out_data.data()),
                 out_data.size() * sizeof(T *), dev_ctx.stream());
    T **out_array_data = reinterpret_cast<T **>(tmp_out_array->ptr());

    const int block = 1024;
    dim3 threads(block, 1);
    dim3 grid(out_data.size(), std::max(batch_size, 1),
              std::max(static_cast<int>(item_dim), 1));

    sequence_pool_all_kernel<T, block><<<grid, threads, 0, dev_ctx.stream()>>>(
        in_array_data, pad_value, lod_array_data, batch_size, item_dim,
        out_array_data);
  }
};

template <typename T>
class SequencePoolAllGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto grad_vars = ctx.MultiInput<Tensor>(framework::GradVarName("Out"));
    auto in_vars = ctx.MultiInput<LoDTensor>("X");
    auto out_vars = ctx.MultiOutput<LoDTensor>(framework::GradVarName("X"));

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    int batch_size = in_vars[0]->lod()[0].size() - 1;
    auto in_num = in_vars.size();
    std::vector<const size_t *> lod_data;
    for (int i = 0; i < in_num; ++i) {
      auto dims = in_vars[i]->dims();
      auto &lod = in_vars[i]->lod();
      PADDLE_ENFORCE_EQ(lod.size(), 1,
                        platform::errors::InvalidArgument(
                            "Currently, it only supports lod_level = 1."));
      PADDLE_ENFORCE_EQ(
          batch_size,
          /*batch size = */ static_cast<int64_t>(lod[0].size() - 1),
          platform::errors::InvalidArgument(
              "There are error of lod mesage in inputs."));
      PADDLE_ENFORCE_GE(
          dims[0],
          /*batch size = */ static_cast<int64_t>(lod[0].size() - 1),
          platform::errors::InvalidArgument("The first dimension of Input(X) "
                                            "must be large than batch size."));
      lod_data.emplace_back(lod[0].CUDAData(dev_ctx.GetPlace()));

      out_vars[i]->mutable_data<T>(ctx.GetPlace());
    }
    const size_t item_dim = grad_vars[0]->numel() / grad_vars[0]->dims()[0];

    std::vector<const T *> in_data;
    std::vector<T *> out_data;
    for (int i = 0; i < in_num; ++i) {
      in_data.emplace_back(grad_vars[i]->data<T>());
      out_data.emplace_back(out_vars[i]->data<T>());
    }

    auto tmp_in_array = memory::Alloc(dev_ctx, in_data.size() * sizeof(T *));
    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_in_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(in_data.data()),
                 in_data.size() * sizeof(T *), dev_ctx.stream());
    T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());

    auto tmp_lod_array =
        memory::Alloc(dev_ctx, lod_data.size() * sizeof(size_t *));
    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_lod_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(lod_data.data()),
                 lod_data.size() * sizeof(size_t *), dev_ctx.stream());
    size_t **lod_array_data = reinterpret_cast<size_t **>(tmp_lod_array->ptr());

    auto tmp_out_array = memory::Alloc(dev_ctx, out_data.size() * sizeof(T *));
    memory::Copy(boost::get<platform::CUDAPlace>(dev_ctx.GetPlace()),
                 tmp_out_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(out_data.data()),
                 out_data.size() * sizeof(T *), dev_ctx.stream());
    T **out_array_data = reinterpret_cast<T **>(tmp_out_array->ptr());

    dim3 threads(std::min(1024, static_cast<int>(item_dim)), 1);
    dim3 grid(out_data.size(), std::max(batch_size, 1));

    sequence_pool_all_grad_kernel<
        T, SumPoolGradFunctor<T>><<<grid, threads, 0, dev_ctx.stream()>>>(
        SumPoolGradFunctor<T>(), in_array_data, lod_array_data, batch_size,
        item_dim, out_array_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using GPUCtx = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(sequence_pool_all,
                        ops::SequencePoolAllCUDAKernel<GPUCtx, float>,
                        ops::SequencePoolAllCUDAKernel<GPUCtx, double>);

REGISTER_OP_CUDA_KERNEL(sequence_pool_all_grad,
                        ops::SequencePoolAllGradOpCUDAKernel<float>,
                        ops::SequencePoolAllGradOpCUDAKernel<double>);
