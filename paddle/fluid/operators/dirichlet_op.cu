#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/operators/dirichlet_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_function.h"
#include "paddle/fluid/operators/reduce_ops/reduce_op.h"
#include "paddle/fluid/operators/reduce_ops/reduce_sum_op.h"
#include "paddle/fluid/platform/for_range.h"

#ifdef PADDLE_WITH_CUDA
#include <hiprand/hiprand_kernel.h>
#endif
#ifdef PADDLE_WITH_HIP
#include <hiprand_kernel.h>
#endif

#if defined(PADDLE_WITH_CUDA)
using COMPAT_RANDSTATEPHILOX4_32_10_T = hiprandStatePhilox4_32_10_t;
#define COMPAT_RAND_INIT hiprand_init
#define COMPAT_RAND_UNIFORM hiprand_uniform
#define COMPAT_RAND_NORMAL hiprand_normal
#elif defined(PADDLE_WITH_HIP)
using COMPAT_RANDSTATEPHILOX4_32_10_T = hiprandStatePhilox4_32_10_t;
#define COMPAT_RAND_INIT hiprand_init
#define COMPAT_RAND_UNIFORM hiprand_uniform
#define COMPAT_RAND_NORMAL hiprand_normal
#endif

namespace paddle {
namespace operators {
template <typename T>
struct GammaCUDAFunctor {
  GammaCUDAFunctor(const T* alpha, T* gamma, uint64_t seed, uint64_t offset)
      : alpha_(alpha), gamma_(gamma), seed_(seed), offset_(offset) {}

  DEVICE void operator()(int64_t index) {
    // hiprand initialization
    COMPAT_RANDSTATEPHILOX4_32_10_T state;
    COMPAT_RAND_INIT(/*seed=*/seed_, /*subsequence=*/index, /*offset=*/offset_,
                     &state);

    // sample
    auto uniform_lambda = [&state]() { return COMPAT_RAND_UNIFORM(&state); };
    BaseSampler<T, decltype(uniform_lambda)> standard_uniform(uniform_lambda);
    auto normal_lambda = [&state]() { return COMPAT_RAND_NORMAL(&state); };
    BaseSampler<T, decltype(normal_lambda)> standard_normal(normal_lambda);

    auto sample =
        sample_gamma<T, T, decltype(uniform_lambda), decltype(normal_lambda)>(
            alpha_[index], standard_uniform, standard_normal);
    gamma_[index] = std::max(std::numeric_limits<T>::min(), sample);
  }

  const T* alpha_;
  T* gamma_;
  const uint64_t seed_;
  const uint64_t offset_;
};

template <typename T>
struct DirichletSampler<platform::CUDADeviceContext, T> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor* alpha, framework::Tensor* out) {
    auto& dev_ctx = ctx.device_context<platform::CUDADeviceContext>();

    // init state, seed & offset for all threads
    int device_id = ctx.GetPlace().GetDeviceId();
    auto p_gen = framework::GetDefaultCUDAGenerator(device_id);
    auto seed_and_offset = p_gen->IncrementOffset(10);  // hard-coded offset
    auto seed = seed_and_offset.first;
    auto offset = seed_and_offset.second;

    // sample from K gamma distributions, where K=alpha.numel()
    framework::Tensor gamma_samples;
    gamma_samples.mutable_data<T>(alpha->dims(), dev_ctx.GetPlace());
    GammaCUDAFunctor<T> gamma_functor(alpha->data<T>(), gamma_samples.data<T>(),
                                      seed, offset);
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx,
                                                              out->numel());
    for_range(gamma_functor);

    // normalize them into a simplex, along the last axis
    framework::Tensor gamma_sum;
    auto new_shape = gamma_samples.dims();
    new_shape[new_shape.size() - 1] = 1;
    gamma_sum.mutable_data<T>(new_shape, dev_ctx.GetPlace());

    ReduceKernelFunctor<platform::CUDADeviceContext, T, SumFunctor>(
        &gamma_samples, &gamma_sum, {new_shape.size() - 1}, true, false, ctx)
        .template apply<T>();
    ElementwiseComputeEx<DivFunctor<T>, platform::CUDADeviceContext, T, T>(
        ctx, &gamma_samples, &gamma_sum, -1, DivFunctor<T>(), out);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    dirichlet, ops::DirichletKernel<paddle::platform::CUDADeviceContext, float>,
    ops::DirichletKernel<paddle::platform::CUDADeviceContext, double>);
