// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/transform.h>
#include "paddle/fluid/operators/cumprod_op.h"
#include "paddle/fluid/operators/math/inclusive_scan.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"

namespace paddle {
namespace operators {

template <typename T>
struct MultiplyFunctor {
  HOSTDEVICE T operator()(T a, T b) const { return a * b; }
};

template <typename T>
class CumprodOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto *x = ctx.Input<framework::Tensor>("X");
    auto *y = ctx.Output<framework::Tensor>("Out");
    auto dim = ctx.Attr<int>("dim");
    size_t outer_dim, mid_dim, inner_dim;
    GetCumprodDimInfo(x->dims(), dim, &outer_dim, &mid_dim, &inner_dim);

    const auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    const auto &dev_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();
    math::InclusiveScan<T, MultiplyFunctor<T>>(
        x_data, y_data, outer_dim, mid_dim, inner_dim, static_cast<T>(1),
        MultiplyFunctor<T>(), /*reverse=*/false, dev_ctx);
  }
};

template <typename T>
struct IsZeroFunctor {
  HOSTDEVICE bool operator()(T x) const { return x == static_cast<T>(0); }
};

template <typename T>
struct CumprodGradFunctorExceptFirstZero {
  HOSTDEVICE CumprodGradFunctorExceptFirstZero(
      const T *x, const T *y, const T *dy_mul_y_reversed_cumsum,
      const uint8_t *zero_mask, size_t mid_dim, size_t inner_dim, T *dx,
      int64_t *first_zero_idx, T *x_filled_one)
      : x_(x),
        y_(y),
        dy_mul_y_reversed_cumsum_(dy_mul_y_reversed_cumsum),
        zero_mask_(zero_mask),
        mid_dim_(mid_dim),
        inner_dim_(inner_dim),
        dx_(dx),
        first_zero_idx_(first_zero_idx),
        x_filled_one_(x_filled_one) {}

  HOSTDEVICE void operator()(size_t idx) const {
    auto inner_idx = idx % inner_dim_;
    auto outer_idx = idx / (mid_dim_ * inner_dim_);
    auto mid_idx = (idx - inner_idx) / inner_dim_ % mid_dim_;
    auto mask = zero_mask_[idx];
    bool should_fill_one = true;

    if (mask == 0) {
      dx_[idx] = dy_mul_y_reversed_cumsum_[idx] / x_[idx];
      if (mid_idx == mid_dim_ - 1) {
        // record first zero position as -1, i.e., no zero
        first_zero_idx_[outer_idx * inner_dim_ + inner_idx] = -1;
      }
    } else if (mid_idx > 0) {                  // mask > 0
      if (zero_mask_[idx - inner_dim_] > 0) {  // not first zero
        dx_[idx] = 0;
        should_fill_one = false;
      } else {
        // idx is the first zero position, it should be recorded
        dx_[idx] = y_[idx - inner_dim_];
        first_zero_idx_[outer_idx * inner_dim_ + inner_idx] = mid_idx;
      }
    } else {  // the first zero position is index 0
      dx_[idx] = 1;
      first_zero_idx_[outer_idx * inner_dim_ + inner_idx] = 0;
    }

    x_filled_one_[idx] = should_fill_one ? 1 : x_[idx];
  }

 private:
  const T *x_;
  const T *y_;
  const T *dy_mul_y_reversed_cumsum_;
  const uint8_t *zero_mask_;
  size_t mid_dim_;
  size_t inner_dim_;
  T *dx_;
  int64_t *first_zero_idx_;
  T *x_filled_one_;
};

template <typename T>
struct FillFirstZeroPositionGradFunctor {
  HOSTDEVICE FillFirstZeroPositionGradFunctor(const int64_t *first_zero_idx,
                                              const T *grad_value,
                                              size_t mid_dim, size_t inner_dim,
                                              T *dx)
      : first_zero_idx_(first_zero_idx),
        grad_value_(grad_value),
        mid_dim_(mid_dim),
        inner_dim_(inner_dim),
        dx_(dx) {}

  HOSTDEVICE void operator()(size_t idx) const {
    auto outer_idx = idx / inner_dim_;
    auto inner_idx = idx % inner_dim_;
    auto mid_idx = first_zero_idx_[idx];
    if (mid_idx >= 0) {
      auto full_idx =
          outer_idx * mid_dim_ * inner_dim_ + mid_idx * inner_dim_ + inner_idx;
      dx_[full_idx] *= grad_value_[full_idx];
    }
  }

 private:
  const int64_t *first_zero_idx_;
  const T *grad_value_;
  size_t mid_dim_;
  size_t inner_dim_;
  T *dx_;
};

/*
Reference to
https://github.com/pytorch/pytorch/blob/master/aten/src/ATen/native/ReduceOps.cpp
input: x, y, dL/dy
output: dL/dx
dL/dx[i] = sum{0<=j<n} (dL/dy[j])*(dy[j]/dx[i]) (1)
         = sum(0<=j<n} (dL/dy[j])*(d(x[0]*x[1]*...*x[j])/dx[i])
if x[i] != 0, dL/dx[i] = sum{i<=j<n} (dL/dy[j])*(y[j]/x[i]) (2)
if x[i] == 0, the formula(2) can not be applied directly.
Suppose k is the first index of zero element, the formula will be:
i > k, dL/dx[i] = 0;
i < k, dL/dx[i] = 1/x[i]*sum{i<=j<n} (dL/dy[j]*y[j])
i = k, dL/dx[i] = y[i-1]*sum{i<=j<n} (dL/dy[j])*(x[i+1]*...*x[j])

First, we will show the main resolution.
We need to judge the relationship between i (current index) and k (index
which corresponds to the first element of 0).
To mark the relationship, we now introduce zero_mask and we also need to
mark the index of the first zero element.
zero_mask = cummax(x[i] == 0);      //label whether x[i]==0 until the index.
zero_index = -1;                    //store the first zero element's index.
e.g. x = [1, 4, 5, 0, 2, 3, 0];
     zero_mask = [0, 0, 0, 1, 1, 1, 1];
     zero_index = 3;
When i < k, we need to calculate the result of sum{i<=j<n}(d_y[j]*y[j]), we can
use reversed cumsum to calculate it.
R = reversed_cumsum(dy[j]*y[j]);     //store the calculation result of the
sum{i<=j<n}(d_y[j]*y[j]) and x[k+1],x[k+2],...,x[j] along the index k+1 ~ j.
When i = k, we need to calculate the result of prod{i<w<j}(x[w]).
To calculate it, we introduce x_filled_one, which fill 1 before x[k+1] along
the index 0 ~ k.
e.g. x = [1, 4, 5, 0, 2, 3, 0];
     x_filled_one = [1, 1, 1, 1, 2, 3, 0];
Thus, we can use cumprod(x_filled_one[j]) to calculate the result of
prod{k<=w<j}(x[w]).

Then, we will show more detailed implementation.
for (int i = 0; i < numel; i++) {
    if (zero_mask[i] == 0) {       //case i < k
        dx[i] = R[i] / x[i];
        x_filled_one[i] = 1;
    } else {
        if (i == 0) {              //case i = k
            dx[i] = 1;
            zero_index = i;
            x_filled_one[i] = 1;
        } else {
            if (zero_mask[i-1] == 0) {    //case i = k
                dx[i] = y[i-1];
                zero_index = i;
                x_filled_one[i] = 1;
            } else {                  //case i > k
                dx[i] = 0;
                x_filled_one[i] = x[i];
            }
        }
    }
}
T = reversed_cumsum(dy[j]*cumprod(x_filled_one[j]));
if (zero_index != -1) {
    dx[zero_index] *= T[zero_index];
}
*/

template <typename T>
class CumprodGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto *x = ctx.Input<framework::Tensor>("X");
    const auto *y = ctx.Input<framework::Tensor>("Out");
    const auto *dy =
        ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto *dx = ctx.Output<framework::Tensor>(framework::GradVarName("X"));
    auto dim = ctx.Attr<int>("dim");

    size_t outer_dim, mid_dim, inner_dim;
    GetCumprodDimInfo(x->dims(), dim, &outer_dim, &mid_dim, &inner_dim);
    if (outer_dim == 0 || mid_dim == 0 || inner_dim == 0) return;

    size_t numel = outer_dim * mid_dim * inner_dim;

    const auto *x_data = x->data<T>();
    const auto *y_data = y->data<T>();
    const auto *dy_data = dy->data<T>();

    auto place = ctx.GetPlace();
    const auto &dev_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();
    auto *dx_data = dx->mutable_data<T>(place);

    // deal with complex
    const T *x_data_deal;
    const T *y_data_deal;
    memory::AllocationPtr x_conj;
    memory::AllocationPtr y_conj;
    if (framework::IsComplex<T>::value) {
      x_conj = memory::Alloc(place, numel * sizeof(T));
      auto *x_data_conj = reinterpret_cast<T *>(x_conj->ptr());
      y_conj = memory::Alloc(place, numel * sizeof(T));
      auto *y_data_conj = reinterpret_cast<T *>(y_conj->ptr());

      platform::ForRange<platform::CUDADeviceContext> for_range_x(dev_ctx,
                                                                  numel);
      phi::funcs::ConjFunctor<T> functor_x(x_data, numel, x_data_conj);
      for_range_x(functor_x);

      platform::ForRange<platform::CUDADeviceContext> for_range_y(dev_ctx,
                                                                  numel);
      phi::funcs::ConjFunctor<T> functor_y(y_data, numel, y_data_conj);
      for_range_y(functor_y);
      x_data_deal = x_data_conj;
      y_data_deal = y_data_conj;
    } else {
      x_data_deal = x_data;
      y_data_deal = y_data;
    }

// Step 1: find cummax-ed zero mask of x
#ifdef PADDLE_WITH_CUDA
    const auto &exec_policy = thrust::cuda::par.on(dev_ctx.stream());
#else
    const auto &exec_policy = thrust::hip::par.on(dev_ctx.stream());
#endif
    auto zero_mask_without_cummax =
        memory::Alloc(place, numel * sizeof(uint8_t));
    auto *zero_mask_without_cummax_data =
        reinterpret_cast<uint8_t *>(zero_mask_without_cummax->ptr());
    thrust::transform(
        exec_policy, thrust::device_pointer_cast(x_data_deal),
        thrust::device_pointer_cast(x_data_deal) + numel,
        thrust::device_pointer_cast(zero_mask_without_cummax_data),
        IsZeroFunctor<T>());

    auto zero_mask = memory::Alloc(place, numel * sizeof(uint8_t));
    auto *zero_mask_data = reinterpret_cast<uint8_t *>(zero_mask->ptr());
    math::InclusiveScan<uint8_t, hipcub::Max>(
        zero_mask_without_cummax_data, zero_mask_data, outer_dim, mid_dim,
        inner_dim, static_cast<uint8_t>(0), hipcub::Max(), /*reverse=*/false,
        dev_ctx);
    zero_mask_without_cummax = nullptr;

    // Step 2: calculate reversed cumsum(dy * y)
    auto dy_mul_y = memory::Alloc(place, numel * sizeof(T));
    auto *dy_mul_y_data = reinterpret_cast<T *>(dy_mul_y->ptr());
    thrust::transform(exec_policy, thrust::device_pointer_cast(dy_data),
                      thrust::device_pointer_cast(dy_data) + numel,
                      thrust::device_pointer_cast(y_data_deal),
                      thrust::device_pointer_cast(dy_mul_y_data),
                      MultiplyFunctor<T>());

    auto dy_mul_y_reversed_cumsum = memory::Alloc(place, numel * sizeof(T));
    auto *dy_mul_y_reversed_cumsum_data =
        reinterpret_cast<T *>(dy_mul_y_reversed_cumsum->ptr());
    math::InclusiveScan<T, hipcub::Sum>(
        dy_mul_y_data, dy_mul_y_reversed_cumsum_data, outer_dim, mid_dim,
        inner_dim, static_cast<T>(0), hipcub::Sum(), /*reverse=*/true, dev_ctx);

    // Step 3: calculate the gradient value except the first zero position.
    // The gradient value of the first zero position is filled with out[idx-1],
    // while the gradient value of the other positions are calculated out
    // completely. This functor also:
    //  (1) find the first zero index, i.e., first_zero_idx_data.
    //  (2) fill x_filled_one, which satifies
    //      x_filled_one[i] = x[i], i > pos
    //      x_filled_one[i] = 1, i <= pos
    auto first_zero_idx =
        memory::Alloc(place, outer_dim * inner_dim * sizeof(int64_t));
    auto *first_zero_idx_data =
        reinterpret_cast<int64_t *>(first_zero_idx->ptr());
    auto *x_filled_one_data = dy_mul_y_data;  // reuse former allocated memory
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx, numel);
    CumprodGradFunctorExceptFirstZero<T> functor_except_first_zero(
        x_data_deal, y_data_deal, dy_mul_y_reversed_cumsum_data, zero_mask_data,
        mid_dim, inner_dim, dx_data, first_zero_idx_data, x_filled_one_data);
    for_range(functor_except_first_zero);

    // Step 4: calculate cumprod of x_filled_one
    auto *x_filled_one_cumprod_data =
        dy_mul_y_reversed_cumsum_data;  // reuse former allocated memory
    math::InclusiveScan<T, MultiplyFunctor<T>>(
        x_filled_one_data, x_filled_one_cumprod_data, outer_dim, mid_dim,
        inner_dim, static_cast<T>(1), MultiplyFunctor<T>(), /*reverse=*/false,
        dev_ctx);

    // Step 5: calculate reversed cumsum(dy * x_filled_one_cumprod)
    auto *dy_mul_x_filled_one_cumprod =
        dy_mul_y_data;  // reuse former allocated memory
    thrust::transform(exec_policy, thrust::device_pointer_cast(dy_data),
                      thrust::device_pointer_cast(dy_data) + numel,
                      thrust::device_pointer_cast(x_filled_one_cumprod_data),
                      thrust::device_pointer_cast(dy_mul_x_filled_one_cumprod),
                      MultiplyFunctor<T>());
    auto *dy_mul_x_filled_one_cumprod_reversed_cumsum =
        dy_mul_y_reversed_cumsum_data;  // reuse former allocated memory
    math::InclusiveScan<T, hipcub::Sum>(
        dy_mul_x_filled_one_cumprod,
        dy_mul_x_filled_one_cumprod_reversed_cumsum, outer_dim, mid_dim,
        inner_dim, static_cast<T>(0), hipcub::Sum(),
        /*reverse=*/true, dev_ctx);

    // Step 6: fill zero pos gradient value
    platform::ForRange<platform::CUDADeviceContext>
        for_range_fill_zero_pos_grad(dev_ctx, outer_dim * inner_dim);
    FillFirstZeroPositionGradFunctor<T> fill_first_zero_pos_grad_functor(
        first_zero_idx_data, dy_mul_x_filled_one_cumprod_reversed_cumsum,
        mid_dim, inner_dim, dx_data);
    for_range_fill_zero_pos_grad(fill_first_zero_pos_grad_functor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    cumprod, ops::CumprodOpCUDAKernel<float>, ops::CumprodOpCUDAKernel<double>,
    ops::CumprodOpCUDAKernel<int>, ops::CumprodOpCUDAKernel<int64_t>,
    ops::CumprodOpCUDAKernel<paddle::platform::complex<float>>,
    ops::CumprodOpCUDAKernel<paddle::platform::complex<double>>);

REGISTER_OP_CUDA_KERNEL(
    cumprod_grad, ops::CumprodGradOpCUDAKernel<float>,
    ops::CumprodGradOpCUDAKernel<double>, ops::CumprodGradOpCUDAKernel<int>,
    ops::CumprodGradOpCUDAKernel<int64_t>,
    ops::CumprodGradOpCUDAKernel<paddle::platform::complex<float>>,
    ops::CumprodGradOpCUDAKernel<paddle::platform::complex<double>>);
