#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/conv_transpose_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/padding.h"
#include "paddle/fluid/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using DataLayout = platform::DataLayout;

static constexpr size_t kConvCUDNNWorkspaceLimitBytes = 1024 * 1024 * 1024;

template <typename T, int D>
static void DataTranspose(const framework::ExecutionContext& ctx,
                          const Tensor* input, Tensor* output,
                          const std::vector<int>& axis, int flag = 0) {
  auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::Transpose<platform::CUDADeviceContext, T, D> transpose;
  auto in_dims = input->dims();
  std::vector<int64_t> input_transpose_vec;
  for (size_t i = 0; i < axis.size(); ++i) {
    if (flag == 0)
      input_transpose_vec.push_back(in_dims[axis[i]]);
    else
      input_transpose_vec.push_back(in_dims[i]);
  }
  framework::DDim input_transpose_dims(
      framework::make_ddim(input_transpose_vec));
  output->mutable_data<T>(input_transpose_dims, ctx.GetPlace());
  transpose(dev_ctx, *input, output, axis);
}

template <typename T>
class CUDNNConvTransposeOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "It must use CUDAPlace.");
    auto* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* output = ctx.Output<Tensor>("Output");

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");

    // cudnn v5 does not support dilations
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    int user_workspace_size = ctx.Attr<int>("workspace_size_MB");
    const T* filter_data = filter->data<T>();
    const std::string data_layout_str = ctx.Attr<std::string>("data_format");
    const paddle::operators::DataLayout data_layout =
        (data_layout_str == "NCHW" ? DataLayout::kNCHW : DataLayout::kNHWC);

    // if channel_last, transpose to channel_first
    Tensor input_transpose;
    std::vector<int> input_vec = framework::vectorize<int>(input->dims());
    std::vector<int> output_vec = framework::vectorize<int>(output->dims());
    if (data_layout == DataLayout::kNHWC) {
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 3, 1, 2};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output->dims()[axis[i]];
        }
        DataTranspose<T, 4>(ctx, input, &input_transpose, axis);
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 4, 1, 2, 3};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output->dims()[axis[i]];
        }
        DataTranspose<T, 5>(ctx, input, &input_transpose, axis);
      }
    } else {
      input_transpose = *input;
    }

    // update padding and dilation
    auto in_dims = input_transpose.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    std::vector<int> input_pad(input_transpose.dims().size() * 2, 0);
    Tensor transformed_input;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = input_transpose.dims()[0];
      new_input_shape_vec[1] = input_transpose.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            input_transpose.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      const int rank = input_transpose.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, input_transpose, pad_value, &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, input_transpose, pad_value, &transformed_input);
        } break;
        default:
          PADDLE_ENFORCE_EQ(
              rank == 4 || rank == 5, true,
              "Op(ConvTranspose) only supports 4-D or 5-D input Tensor.");
      }
    } else {
      transformed_input = input_transpose;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    std::vector<int64_t> starts(data_dim, 0);
    std::vector<int64_t> ends(data_dim, 0);
    std::vector<int64_t> axes(data_dim, 0);
    for (size_t i = 0; i < data_dim; ++i) {
      starts[i] = input_pad[2 * i + 4] * (strides[i] + 1);
      ends[i] = starts[i] + output_vec[i + 2];
      axes[i] = i + 2;
    }

    const T* input_data = transformed_input.data<T>();
    input_vec = framework::vectorize<int>(transformed_input.dims());

    std::vector<int> transformed_output_vec = output_vec;
    for (size_t i = 0; i < data_dim; ++i) {
      transformed_output_vec[i + 2] =
          output_vec[i + 2] +
          (input_pad[2 * i + 4] + input_pad[2 * i + 5]) * strides[i] -
          2 * padding_common[i] + paddings[2 * i] + paddings[2 * i + 1];
    }

    Tensor transformed_output;
    if (!is_sys_pad) {
      DDim transformed_output_shape(
          framework::make_ddim(transformed_output_vec));
      transformed_output.mutable_data<T>(transformed_output_shape,
                                         ctx.GetPlace());
    } else {
      output->mutable_data<T>(ctx.GetPlace());
      transformed_output.ShareDataWith(*output);
      transformed_output.Resize(framework::make_ddim(transformed_output_vec));
    }
    T* transformed_output_data = transformed_output.data<T>();

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedFilterDescriptor filter_desc;
    ScopedConvolutionDescriptor conv_desc;
    DataLayout layout;

    if (strides.size() == 2U) {
      layout = DataLayout::kNCHW;
    } else {
      layout = DataLayout::kNCDHW;
    }

    // (N, M, H, W) or (N, M, D, H, W)
    hipdnnTensorDescriptor_t cudnn_input_desc =
        input_desc.descriptor<T>(layout, input_vec, groups);
    // (N, C, O_h, O_w) or (N, C, O_d, O_h, O_w)
    hipdnnTensorDescriptor_t cudnn_output_desc =
        output_desc.descriptor<T>(layout, transformed_output_vec, groups);
    // (M, C, K_h, K_w) or (M, C, K_d, K_h, K_w)
    hipdnnFilterDescriptor_t cudnn_filter_desc = filter_desc.descriptor<T>(
        layout, framework::vectorize<int>(filter->dims()), groups);
    hipdnnConvolutionDescriptor_t cudnn_conv_desc =
        conv_desc.descriptor<T>(padding_common, strides, dilations);

    // ------------------- cudnn conv workspace ---------------------
    size_t workspace_size_in_bytes;  // final workspace to allocate.
    size_t workspace_size_limit = kConvCUDNNWorkspaceLimitBytes;
    if (user_workspace_size > 0) {
      workspace_size_limit = user_workspace_size * 1024 * 1024;
    }
    // ------------------- cudnn conv algorithm ---------------------
    hipdnnConvolutionBwdDataAlgo_t algo;
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    // Get the algorithm
    CUDNN_ENFORCE(platform::dynload::hipdnnGetConvolutionBackwardDataAlgorithm(
        handle, cudnn_filter_desc, cudnn_input_desc, cudnn_conv_desc,
        // dxDesc: Handle to the previously initialized output tensor
        // descriptor.
        cudnn_output_desc, HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT,
        workspace_size_limit, &algo));

    // get workspace size able to allocate
    CUDNN_ENFORCE(
        platform::dynload::hipdnnGetConvolutionBackwardDataWorkspaceSize(
            handle, cudnn_filter_desc, cudnn_input_desc, cudnn_conv_desc,
            cudnn_output_desc, algo, &workspace_size_in_bytes));

    // ------------------- cudnn conv transpose forward ---------------------
    int input_offset =
        transformed_input.numel() / transformed_input.dims()[0] / groups;
    int output_offset =
        transformed_output.numel() / transformed_output.dims()[0] / groups;
    int filter_offset = filter->numel() / groups;
    T alpha = 1.0f, beta = 0.0f;
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    for (int g = 0; g < groups; g++) {
      auto cudnn_func = [&](void* cudnn_workspace) {
        CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionBackwardData(
            handle, &alpha, cudnn_filter_desc, filter_data + filter_offset * g,
            cudnn_input_desc, input_data + input_offset * g, cudnn_conv_desc,
            algo, cudnn_workspace, workspace_size_in_bytes, &beta,
            cudnn_output_desc, transformed_output_data + output_offset * g));
      };
      workspace_handle.RunFunc(cudnn_func, workspace_size_in_bytes);
    }

    if (!is_sys_pad && strides.size() == 2U) {
      Slice<paddle::platform::CUDADeviceContext, T, 4>(
          ctx, &transformed_output, output, starts, ends, axes);
    } else if (!is_sys_pad && strides.size() == 3U) {
      Slice<paddle::platform::CUDADeviceContext, T, 5>(
          ctx, &transformed_output, output, starts, ends, axes);
    }

    if (data_layout == DataLayout::kNHWC) {
      Tensor output_transpose;
      Tensor output_nchw;
      output_nchw.ShareDataWith(*output);
      output_nchw.Resize(framework::make_ddim(output_vec));
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 2, 3, 1};
        DataTranspose<T, 4>(ctx, &output_nchw, &output_transpose, axis);
        *output = output_transpose;
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 2, 3, 4, 1};
        DataTranspose<T, 5>(ctx, &output_nchw, &output_transpose, axis);
        *output = output_transpose;
      }
    }
  }
};

template <typename T>
class CUDNNConvTransposeGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto input = ctx.Input<Tensor>("Input");
    auto filter = ctx.Input<Tensor>("Filter");
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));
    const T* filter_data = filter->data<T>();

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    // cudnn v5 does not support dilations
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    int user_workspace_size = ctx.Attr<int>("workspace_size_MB");
    const std::string data_layout_str = ctx.Attr<std::string>("data_format");
    const paddle::operators::DataLayout data_layout =
        (data_layout_str == "NCHW" ? DataLayout::kNCHW : DataLayout::kNHWC);

    // if channel_last, transpose to channel_first
    Tensor input_transpose;
    Tensor output_grad_transpose;
    std::vector<int> input_vec = framework::vectorize<int>(input->dims());
    std::vector<int> output_vec =
        framework::vectorize<int>(output_grad->dims());
    if (data_layout == DataLayout::kNHWC) {
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 3, 1, 2};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output_grad->dims()[axis[i]];
        }
        DataTranspose<T, 4>(ctx, input, &input_transpose, axis);
        DataTranspose<T, 4>(ctx, output_grad, &output_grad_transpose, axis);
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 4, 1, 2, 3};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output_grad->dims()[axis[i]];
        }
        DataTranspose<T, 5>(ctx, input, &input_transpose, axis);
        DataTranspose<T, 5>(ctx, output_grad, &output_grad_transpose, axis);
      }
    } else {
      input_transpose = *input;
      output_grad_transpose = *output_grad;
    }

    // update padding and dilation
    auto in_dims = input_transpose.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    std::vector<int> input_pad(input_transpose.dims().size() * 2, 0);
    Tensor transformed_output_grad;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_output_grad_shape_vec(data_dim + 2);
      new_output_grad_shape_vec[0] = output_grad_transpose.dims()[0];
      new_output_grad_shape_vec[1] = output_grad_transpose.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_output_grad_shape_vec[i + 2] =
            output_grad_transpose.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_output_grad_shape(
          framework::make_ddim(new_output_grad_shape_vec));
      transformed_output_grad.Resize(new_output_grad_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_output_grad =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_output_grad_shape, dev_ctx);
      const int rank = input_transpose.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, output_grad_transpose, pad_value,
              &transformed_output_grad);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, output_grad_transpose, pad_value,
              &transformed_output_grad);
        } break;
        default:
          PADDLE_ENFORCE_EQ(
              rank == 4 || rank == 5, true,
              "Op(ConvTranspose) only supports 4-D or 5-D input Tensor.");
      }
    } else {
      transformed_output_grad = output_grad_transpose;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = input_transpose.data<T>();
    const T* output_grad_data = transformed_output_grad.data<T>();
    output_vec = framework::vectorize<int>(transformed_output_grad.dims());

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedFilterDescriptor filter_desc;
    ScopedConvolutionDescriptor conv_desc;
    DataLayout layout;

    if (strides.size() == 2U) {
      layout = DataLayout::kNCHW;
    } else {
      layout = DataLayout::kNCDHW;
    }

    // Input: (N, M, H, W) or (N, M, D, H, W)
    hipdnnTensorDescriptor_t cudnn_input_desc =
        input_desc.descriptor<T>(layout, input_vec, groups);
    // Output: (N, C, O_h, O_w) or (N, C, O_d, O_h, O_w)
    hipdnnTensorDescriptor_t cudnn_output_desc =
        output_desc.descriptor<T>(layout, output_vec, groups);
    // Filter (M, C, K_h, K_w) or (M, C, K_d K_h, K_w)
    hipdnnFilterDescriptor_t cudnn_filter_desc = filter_desc.descriptor<T>(
        layout, framework::vectorize<int>(filter->dims()), groups);

    hipdnnConvolutionDescriptor_t cudnn_conv_desc =
        conv_desc.descriptor<T>(padding_common, strides, dilations);

    // ------------------- cudnn backward algorithm ---------------------
    hipdnnConvolutionFwdAlgo_t data_algo;
    hipdnnConvolutionBwdFilterAlgo_t filter_algo;
    size_t bwd_filter_ws_size, fwd_ws_size;
    size_t workspace_size_in_bytes = 0;
    size_t workspace_size_limit = kConvCUDNNWorkspaceLimitBytes;
    if (user_workspace_size > 0) {
      workspace_size_limit = user_workspace_size * 1024 * 1024;
    }

    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    if (input_grad) {
      // choose backward algorithm for data
      CUDNN_ENFORCE(platform::dynload::hipdnnGetConvolutionForwardAlgorithm(
          handle, cudnn_output_desc, cudnn_filter_desc, cudnn_conv_desc,
          cudnn_input_desc, HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
          workspace_size_limit, &data_algo));
      CUDNN_ENFORCE(platform::dynload::hipdnnGetConvolutionForwardWorkspaceSize(
          handle, cudnn_output_desc, cudnn_filter_desc, cudnn_conv_desc,
          cudnn_input_desc, data_algo, &fwd_ws_size));
      workspace_size_in_bytes = std::max(workspace_size_in_bytes, fwd_ws_size);
    }

    if (filter_grad) {
      // choose backward algorithm for filter
      CUDNN_ENFORCE(
          platform::dynload::hipdnnGetConvolutionBackwardFilterAlgorithm(
              handle, cudnn_output_desc, cudnn_input_desc, cudnn_conv_desc,
              cudnn_filter_desc,
              HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT,
              workspace_size_limit, &filter_algo));

      // get workspace for backwards filter algorithm
      CUDNN_ENFORCE(
          platform::dynload::hipdnnGetConvolutionBackwardFilterWorkspaceSize(
              handle, cudnn_output_desc, cudnn_input_desc, cudnn_conv_desc,
              cudnn_filter_desc, filter_algo, &bwd_filter_ws_size));
      workspace_size_in_bytes =
          std::max(workspace_size_in_bytes, bwd_filter_ws_size);
    }

    // ------------------- cudnn conv backward data ---------------------
    // FIXME(typhoonzero): template type T may not be the same as cudnn call.
    int input_offset = input->numel() / input->dims()[0] / groups;
    int output_grad_offset = transformed_output_grad.numel() /
                             transformed_output_grad.dims()[0] / groups;
    int filter_offset = filter->numel() / groups;
    T alpha = 1.0f, beta = 0.0f;
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    if (input_grad) {
      T* input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
      // Because beta is zero, it is unnecessary to reset input_grad.
      for (int g = 0; g < groups; g++) {
        auto cudnn_func = [&](void* cudnn_workspace) {
          CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionForward(
              handle, &alpha, cudnn_output_desc,
              output_grad_data + output_grad_offset * g, cudnn_filter_desc,
              filter_data + filter_offset * g, cudnn_conv_desc, data_algo,
              cudnn_workspace, workspace_size_in_bytes, &beta, cudnn_input_desc,
              input_grad_data + input_offset * g));
        };
        workspace_handle.RunFunc(cudnn_func, workspace_size_in_bytes);
      }

      if (data_layout == DataLayout::kNHWC) {
        Tensor input_grad_transpose;
        Tensor input_grad_nchw;
        input_grad_nchw.ShareDataWith(*input_grad);
        input_grad_nchw.Resize(framework::make_ddim(input_vec));
        if (strides.size() == 2U) {
          std::vector<int> axis = {0, 2, 3, 1};
          DataTranspose<T, 4>(ctx, &input_grad_nchw, &input_grad_transpose,
                              axis);
          *input_grad = input_grad_transpose;
        } else if (strides.size() == 3U) {
          std::vector<int> axis = {0, 2, 3, 4, 1};
          DataTranspose<T, 5>(ctx, &input_grad_nchw, &input_grad_transpose,
                              axis);
          *input_grad = input_grad_transpose;
        }
      }
    }

    // ------------------- cudnn conv backward filter ---------------------
    if (filter_grad) {
      T* filter_grad_data = filter_grad->mutable_data<T>(ctx.GetPlace());
      // Because beta is zero, it is unnecessary to reset filter_grad.
      // Gradient with respect to the filter
      for (int g = 0; g < groups; g++) {
        auto cudnn_func = [&](void* cudnn_workspace) {
          CUDNN_ENFORCE(platform::dynload::hipdnnConvolutionBackwardFilter(
              handle, &alpha, cudnn_output_desc,
              output_grad_data + output_grad_offset * g, cudnn_input_desc,
              input_data + input_offset * g, cudnn_conv_desc, filter_algo,
              cudnn_workspace, workspace_size_in_bytes, &beta,
              cudnn_filter_desc, filter_grad_data + filter_offset * g));
        };
        workspace_handle.RunFunc(cudnn_func, workspace_size_in_bytes);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_KERNEL(conv2d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<float>,
                   ops::CUDNNConvTransposeOpKernel<double>);
REGISTER_OP_KERNEL(conv2d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<float>,
                   ops::CUDNNConvTransposeGradOpKernel<double>);

REGISTER_OP_KERNEL(conv3d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<float>,
                   ops::CUDNNConvTransposeOpKernel<double>);
REGISTER_OP_KERNEL(conv3d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<float>,
                   ops::CUDNNConvTransposeGradOpKernel<double>);
