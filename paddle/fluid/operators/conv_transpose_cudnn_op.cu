#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#include "paddle/fluid/operators/conv_transpose_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/padding.h"
#include "paddle/fluid/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = platform::DataLayout;

template <typename T, int D>
static void DataTranspose(const framework::ExecutionContext& ctx,
                          const Tensor* input, Tensor* output,
                          const std::vector<int>& axis, int flag = 0) {
  auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::Transpose<platform::CUDADeviceContext, T, D> transpose;
  auto in_dims = input->dims();
  std::vector<int64_t> input_transpose_vec;
  for (size_t i = 0; i < axis.size(); ++i) {
    if (flag == 0)
      input_transpose_vec.push_back(in_dims[axis[i]]);
    else
      input_transpose_vec.push_back(in_dims[i]);
  }
  framework::DDim input_transpose_dims(
      framework::make_ddim(input_transpose_vec));
  output->mutable_data<T>(input_transpose_dims, ctx.GetPlace());
  transpose(dev_ctx, *input, output, axis);
}

template <typename T>
class CUDNNConvTransposeOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "It must use CUDAPlace.");
    auto* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* output = ctx.Output<Tensor>("Output");

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");

    // cudnn v5 does not support dilations
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    const T* filter_data = filter->data<T>();
    const std::string data_layout_str = ctx.Attr<std::string>("data_format");
    const paddle::operators::DataLayout data_layout =
        (data_layout_str != "NHWC" ? DataLayout::kNCHW : DataLayout::kNHWC);

    // if channel_last, transpose to channel_first
    Tensor input_transpose;
    std::vector<int> input_vec = framework::vectorize<int>(input->dims());
    std::vector<int> output_vec = framework::vectorize<int>(output->dims());
    if (data_layout == DataLayout::kNHWC) {
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 3, 1, 2};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output->dims()[axis[i]];
        }
        DataTranspose<T, 4>(ctx, input, &input_transpose, axis);
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 4, 1, 2, 3};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output->dims()[axis[i]];
        }
        DataTranspose<T, 5>(ctx, input, &input_transpose, axis);
      }
    } else {
      input_transpose = *input;
    }

    // update padding and dilation
    auto in_dims = input_transpose.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    std::vector<int> input_pad(input_transpose.dims().size() * 2, 0);
    Tensor transformed_input;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = input_transpose.dims()[0];
      new_input_shape_vec[1] = input_transpose.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            input_transpose.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      const int rank = input_transpose.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, input_transpose, pad_value, &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, input_transpose, pad_value, &transformed_input);
        } break;
        default:
          PADDLE_ENFORCE_EQ(
              rank == 4 || rank == 5, true,
              "Op(ConvTranspose) only supports 4-D or 5-D input Tensor.");
      }
    } else {
      transformed_input = input_transpose;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    std::vector<int64_t> starts(data_dim, 0);
    std::vector<int64_t> ends(data_dim, 0);
    std::vector<int64_t> axes(data_dim, 0);
    for (size_t i = 0; i < data_dim; ++i) {
      starts[i] = input_pad[2 * i + 4] * (strides[i] + 1);
      ends[i] = starts[i] + output_vec[i + 2];
      axes[i] = i + 2;
    }

    const T* input_data = transformed_input.data<T>();
    input_vec = framework::vectorize<int>(transformed_input.dims());

    std::vector<int> transformed_output_vec = output_vec;
    for (size_t i = 0; i < data_dim; ++i) {
      transformed_output_vec[i + 2] =
          output_vec[i + 2] +
          (input_pad[2 * i + 4] + input_pad[2 * i + 5]) * strides[i] -
          2 * padding_common[i] + paddings[2 * i] + paddings[2 * i + 1];
    }

    Tensor transformed_output;
    if (!is_sys_pad) {
      DDim transformed_output_shape(
          framework::make_ddim(transformed_output_vec));
      transformed_output.mutable_data<T>(transformed_output_shape,
                                         ctx.GetPlace());
    } else {
      output->mutable_data<T>(ctx.GetPlace());
      transformed_output.ShareDataWith(*output);
      transformed_output.Resize(framework::make_ddim(transformed_output_vec));
    }
    T* transformed_output_data = transformed_output.data<T>();

    DataLayout layout;

    int iwo_groups = groups;
    int c_groups = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_groups = 1;
    c_groups = groups;
    groups = 1;
#endif

    if (strides.size() == 2U) {
      layout = DataLayout::kNCHW;
    } else {
      layout = DataLayout::kNCDHW;
    }

    size_t workspace_size = 0;
    hipdnnConvolutionBwdDataAlgo_t algo{};
    // ------------------- cudnn conv algorithm ---------------------
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    auto layout_tensor = GetCudnnTensorFormat(layout);
    bool deterministic = FLAGS_cudnn_deterministic;

    auto dtype = platform::CudnnDataType<T>::type;
    // ------------------- cudnn descriptors ---------------------
    ConvArgs args{&transformed_output,
                  filter,
                  &transformed_input,
                  strides,
                  padding_common,
                  dilations,
                  dtype};
    args.handle = handle;
    args.idesc.set(transformed_output, iwo_groups);
    args.wdesc.set(*filter, layout_tensor, iwo_groups);
    args.odesc.set(transformed_input, iwo_groups);
    args.cdesc.set(dtype, padding_common, strides, dilations, c_groups);

    using search = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
    algo = search::Find<T>(args, false, deterministic, ctx);
    workspace_size =
        std::max(workspace_size, search::GetWorkspaceSize(args, algo));

    // ------------------- cudnn conv transpose forward ---------------------
    int input_offset =
        transformed_input.numel() / transformed_input.dims()[0] / groups;
    int output_offset =
        transformed_output.numel() / transformed_output.dims()[0] / groups;
    int filter_offset = filter->numel() / groups;
    T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    for (int g = 0; g < groups; g++) {
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::hipdnnConvolutionBackwardData(
                handle, &alpha, args.wdesc.desc(),
                filter_data + filter_offset * g, args.odesc.desc(),
                input_data + input_offset * g, args.cdesc.desc(), algo,
                cudnn_workspace, workspace_size, &beta, args.idesc.desc(),
                transformed_output_data + output_offset * g));
      };
      workspace_handle.RunFunc(cudnn_func, workspace_size);
    }
    if (!is_sys_pad && strides.size() == 2U) {
      Slice<paddle::platform::CUDADeviceContext, T, 4>(
          ctx, &transformed_output, output, starts, ends, axes);
    } else if (!is_sys_pad && strides.size() == 3U) {
      Slice<paddle::platform::CUDADeviceContext, T, 5>(
          ctx, &transformed_output, output, starts, ends, axes);
    }

    if (data_layout == DataLayout::kNHWC) {
      Tensor output_transpose;
      Tensor output_nchw;
      output_nchw.ShareDataWith(*output);
      output_nchw.Resize(framework::make_ddim(output_vec));
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 2, 3, 1};
        DataTranspose<T, 4>(ctx, &output_nchw, &output_transpose, axis);
        *output = output_transpose;
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 2, 3, 4, 1};
        DataTranspose<T, 5>(ctx, &output_nchw, &output_transpose, axis);
        *output = output_transpose;
      }
    }
  }
};

template <typename T>
class CUDNNConvTransposeGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto input = ctx.Input<Tensor>("Input");
    auto filter = ctx.Input<Tensor>("Filter");
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));
    const T* filter_data = filter->data<T>();

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    // cudnn v5 does not support dilations
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    int user_workspace_size = ctx.Attr<int>("workspace_size_MB");
    const std::string data_layout_str = ctx.Attr<std::string>("data_format");
    const paddle::operators::DataLayout data_layout =
        (data_layout_str != "NHWC" ? DataLayout::kNCHW : DataLayout::kNHWC);

    // if channel_last, transpose to channel_first
    Tensor input_transpose;
    Tensor output_grad_transpose;
    std::vector<int> input_vec = framework::vectorize<int>(input->dims());
    std::vector<int> output_vec =
        framework::vectorize<int>(output_grad->dims());
    if (data_layout == DataLayout::kNHWC) {
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 3, 1, 2};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output_grad->dims()[axis[i]];
        }
        DataTranspose<T, 4>(ctx, input, &input_transpose, axis);
        DataTranspose<T, 4>(ctx, output_grad, &output_grad_transpose, axis);
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 4, 1, 2, 3};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output_grad->dims()[axis[i]];
        }
        DataTranspose<T, 5>(ctx, input, &input_transpose, axis);
        DataTranspose<T, 5>(ctx, output_grad, &output_grad_transpose, axis);
      }
    } else {
      input_transpose = *input;
      output_grad_transpose = *output_grad;
    }

    // update padding and dilation
    auto in_dims = input_transpose.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    std::vector<int> input_pad(input_transpose.dims().size() * 2, 0);
    Tensor transformed_output_grad;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_output_grad_shape_vec(data_dim + 2);
      new_output_grad_shape_vec[0] = output_grad_transpose.dims()[0];
      new_output_grad_shape_vec[1] = output_grad_transpose.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_output_grad_shape_vec[i + 2] =
            output_grad_transpose.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_output_grad_shape(
          framework::make_ddim(new_output_grad_shape_vec));
      transformed_output_grad.Resize(new_output_grad_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_output_grad =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_output_grad_shape, dev_ctx);
      const int rank = input_transpose.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, output_grad_transpose, pad_value,
              &transformed_output_grad);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, output_grad_transpose, pad_value,
              &transformed_output_grad);
        } break;
        default:
          PADDLE_ENFORCE_EQ(
              rank == 4 || rank == 5, true,
              "Op(ConvTranspose) only supports 4-D or 5-D input Tensor.");
      }
    } else {
      transformed_output_grad = output_grad_transpose;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = input_transpose.data<T>();
    const T* output_grad_data = transformed_output_grad.data<T>();
    output_vec = framework::vectorize<int>(transformed_output_grad.dims());

    // ------------------- cudnn descriptors ---------------------
    DataLayout layout;

    if (strides.size() == 2U) {
      layout = DataLayout::kNCHW;
    } else {
      layout = DataLayout::kNCDHW;
    }

    int iwo_groups = groups;
    int c_groups = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_groups = 1;
    c_groups = groups;
    groups = 1;
#endif

    auto dtype = platform::CudnnDataType<T>::type;

    ConvArgs args1{&transformed_output_grad,
                   filter,
                   &input_transpose,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args2{&transformed_output_grad,
                   filter,
                   &input_transpose,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    hipdnnConvolutionFwdAlgo_t data_algo{};
    hipdnnConvolutionBwdFilterAlgo_t filter_algo{};

    auto layout_tensor = GetCudnnTensorFormat(layout);
    size_t workspace_size = 0;
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    bool deterministic = FLAGS_cudnn_deterministic;
    T* input_grad_data = nullptr;
    T* filter_grad_data = nullptr;
    if (input_grad)
      input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
    if (filter_grad)
      filter_grad_data = filter_grad->mutable_data<T>(ctx.GetPlace());

    if (input_grad) {
      input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
      args1.handle = handle;
      args1.idesc.set(transformed_output_grad, iwo_groups);
      args1.wdesc.set(*filter, layout_tensor, iwo_groups);
      args1.odesc.set(input_transpose, iwo_groups);
      args1.cdesc.set(dtype, padding_common, strides, dilations, c_groups);
      using search1 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
      data_algo = search1::Find<T>(args1, false, deterministic, ctx);
      workspace_size =
          std::max(workspace_size, search1::GetWorkspaceSize(args1, data_algo));
    }

    if (filter_grad) {
      filter_grad_data = filter_grad->mutable_data<T>(ctx.GetPlace());
      args2.handle = handle;
      args2.idesc.set(transformed_output_grad, iwo_groups);
      args2.wdesc.set(*filter_grad, layout_tensor, iwo_groups);
      args2.odesc.set(input_transpose, iwo_groups);
      args2.cdesc.set(dtype, padding_common, strides, dilations, c_groups);
      using search2 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
      filter_algo = search2::Find<T>(args2, false, deterministic, ctx);
      workspace_size = std::max(workspace_size,
                                search2::GetWorkspaceSize(args2, filter_algo));
    }

    // ------------------- cudnn conv backward data ---------------------
    // FIXME(typhoonzero): template type T may not be the same as cudnn call.
    int input_offset = input->numel() / input->dims()[0] / groups;
    int output_grad_offset = transformed_output_grad.numel() /
                             transformed_output_grad.dims()[0] / groups;
    int filter_offset = filter->numel() / groups;
    T alpha = static_cast<T>(1.0), beta = static_cast<T>(0.0);
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    if (input_grad) {
      // Because beta is zero, it is unnecessary to reset input_grad.
      for (int g = 0; g < groups; g++) {
        auto cudnn_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnConvolutionForward(
                  handle, &alpha, args1.idesc.desc(),
                  output_grad_data + output_grad_offset * g, args1.wdesc.desc(),
                  filter_data + filter_offset * g, args1.cdesc.desc(),
                  data_algo, cudnn_workspace, workspace_size, &beta,
                  args1.odesc.desc(), input_grad_data + input_offset * g));
        };
        workspace_handle.RunFunc(cudnn_func, workspace_size);
      }

      if (data_layout == DataLayout::kNHWC) {
        Tensor input_grad_transpose;
        Tensor input_grad_nchw;
        input_grad_nchw.ShareDataWith(*input_grad);
        input_grad_nchw.Resize(framework::make_ddim(input_vec));
        if (strides.size() == 2U) {
          std::vector<int> axis = {0, 2, 3, 1};
          DataTranspose<T, 4>(ctx, &input_grad_nchw, &input_grad_transpose,
                              axis);
          *input_grad = input_grad_transpose;
        } else if (strides.size() == 3U) {
          std::vector<int> axis = {0, 2, 3, 4, 1};
          DataTranspose<T, 5>(ctx, &input_grad_nchw, &input_grad_transpose,
                              axis);
          *input_grad = input_grad_transpose;
        }
      }
    }

    // ------------------- cudnn conv backward filter ---------------------
    if (filter_grad) {
      // Because beta is zero, it is unnecessary to reset filter_grad.
      // Gradient with respect to the filter
      for (int g = 0; g < groups; g++) {
        auto cudnn_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnConvolutionBackwardFilter(
                  handle, &alpha, args2.idesc.desc(),
                  output_grad_data + output_grad_offset * g, args2.odesc.desc(),
                  input_data + input_offset * g, args2.cdesc.desc(),
                  filter_algo, cudnn_workspace, workspace_size, &beta,
                  args2.wdesc.desc(), filter_grad_data + filter_offset * g));
        };
        workspace_handle.RunFunc(cudnn_func, workspace_size);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_KERNEL(conv2d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeOpKernel<float>,
                   ops::CUDNNConvTransposeOpKernel<double>);
REGISTER_OP_KERNEL(conv2d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeGradOpKernel<float>,
                   ops::CUDNNConvTransposeGradOpKernel<double>);

REGISTER_OP_KERNEL(conv3d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeOpKernel<float>,
                   ops::CUDNNConvTransposeOpKernel<double>);
REGISTER_OP_KERNEL(conv3d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeGradOpKernel<float>,
                   ops::CUDNNConvTransposeGradOpKernel<double>);
