#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memory.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/conv_miopen_helper.h"
#else
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#endif
#include "paddle/fluid/operators/conv_transpose_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/padding.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T, int D>
static void DataTranspose(const framework::ExecutionContext& ctx,
                          const Tensor* input, Tensor* output,
                          const std::vector<int>& axis, int flag = 0) {
  auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::Transpose<platform::CUDADeviceContext, T, D> transpose;
  auto in_dims = input->dims();
  std::vector<int64_t> input_transpose_vec;
  for (size_t i = 0; i < axis.size(); ++i) {
    if (flag == 0)
      input_transpose_vec.push_back(in_dims[axis[i]]);
    else
      input_transpose_vec.push_back(in_dims[i]);
  }
  framework::DDim input_transpose_dims(
      framework::make_ddim(input_transpose_vec));
  output->mutable_data<T>(input_transpose_dims, ctx.GetPlace());
  transpose(dev_ctx, *input, output, axis);
}

template <typename T>
class CUDNNConvTransposeOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        paddle::platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    auto* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* output = ctx.Output<Tensor>("Output");

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");

    // cudnn v5 does not support dilations
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    const T* filter_data = filter->data<T>();
    const std::string data_layout_str = ctx.Attr<std::string>("data_format");
    const paddle::platform::DataLayout data_layout =
        (data_layout_str != "NHWC" ? platform::DataLayout::kNCHW
                                   : platform::DataLayout::kNHWC);

    // if channel_last, transpose to channel_first
    Tensor input_transpose;
    std::vector<int> input_vec = framework::vectorize<int>(input->dims());
    std::vector<int> output_vec = framework::vectorize<int>(output->dims());
    if (data_layout == platform::DataLayout::kNHWC) {
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 3, 1, 2};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output->dims()[axis[i]];
        }
        DataTranspose<T, 4>(ctx, input, &input_transpose, axis);
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 4, 1, 2, 3};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output->dims()[axis[i]];
        }
        DataTranspose<T, 5>(ctx, input, &input_transpose, axis);
      }
    } else {
      input_transpose = *input;
    }

    // update padding and dilation
    auto in_dims = input_transpose.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    std::vector<int> input_pad(input_transpose.dims().size() * 2, 0);
    Tensor transformed_input;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = input_transpose.dims()[0];
      new_input_shape_vec[1] = input_transpose.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            input_transpose.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      const int rank = input_transpose.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, input_transpose, pad_value, &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, input_transpose, pad_value, &transformed_input);
        } break;
        default:
          PADDLE_THROW(platform::errors::InvalidArgument(
              "Op(ConvTranspose) only supports 4-D or 5-D input Tensor."));
      }
    } else {
      transformed_input = input_transpose;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    std::vector<int64_t> starts(data_dim, 0);
    std::vector<int64_t> ends(data_dim, 0);
    std::vector<int64_t> axes(data_dim, 0);
    for (size_t i = 0; i < data_dim; ++i) {
      starts[i] = input_pad[2 * i + 4] * (strides[i] + 1);
      ends[i] = starts[i] + output_vec[i + 2];
      axes[i] = i + 2;
    }

    const T* input_data = transformed_input.data<T>();
    input_vec = framework::vectorize<int>(transformed_input.dims());

    std::vector<int> transformed_output_vec = output_vec;
    for (size_t i = 0; i < data_dim; ++i) {
      transformed_output_vec[i + 2] =
          output_vec[i + 2] +
          (input_pad[2 * i + 4] + input_pad[2 * i + 5]) * strides[i] -
          2 * padding_common[i] + paddings[2 * i] + paddings[2 * i + 1];
    }

    Tensor transformed_output;
    if (!is_sys_pad) {
      DDim transformed_output_shape(
          framework::make_ddim(transformed_output_vec));
      transformed_output.mutable_data<T>(transformed_output_shape,
                                         ctx.GetPlace());
    } else {
      output->mutable_data<T>(ctx.GetPlace());
      transformed_output.ShareDataWith(*output);
      transformed_output.Resize(framework::make_ddim(transformed_output_vec));
    }
    T* transformed_output_data = transformed_output.data<T>();

    platform::DataLayout layout;

    int iwo_groups = groups;
    int c_groups = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_groups = 1;
    c_groups = groups;
    groups = 1;
#endif

    if (strides.size() == 2U) {
      layout = platform::DataLayout::kNCHW;
    } else {
      layout = platform::DataLayout::kNCDHW;
    }

    size_t workspace_size = 0;
#ifdef PADDLE_WITH_HIP
    miopenConvBwdDataAlgorithm_t algo{};
#else
    hipdnnConvolutionBwdDataAlgo_t algo{};
#endif
    // ------------------- cudnn conv algorithm ---------------------
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    auto layout_tensor = GetCudnnTensorFormat(layout);
    bool deterministic = FLAGS_cudnn_deterministic;

    auto dtype = platform::CudnnDataType<T>::type;
    // ------------------- cudnn descriptors ---------------------
    ConvArgs args{&transformed_output,
                  filter,
                  &transformed_input,
                  strides,
                  padding_common,
                  dilations,
                  dtype};
    args.handle = handle;
    args.idesc.set(transformed_output, iwo_groups);
    args.wdesc.set(*filter, layout_tensor, iwo_groups);
    args.odesc.set(transformed_input, iwo_groups);
    args.cdesc.set(dtype, padding_common, strides, dilations,
                   platform::AllowTF32Cudnn(), c_groups);

#ifdef PADDLE_WITH_HIP
    using search = SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
#else
    using search = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
#endif

    algo = search::Find<T>(args, false, deterministic, ctx);
    workspace_size =
        std::max(workspace_size, search::GetWorkspaceSize(args, algo));

    // ------------------- cudnn conv transpose forward ---------------------
    int input_offset =
        transformed_input.numel() / transformed_input.dims()[0] / groups;
    int output_offset =
        transformed_output.numel() / transformed_output.dims()[0] / groups;
    int filter_offset = filter->numel() / groups;
    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = 0.0f;
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    for (int g = 0; g < groups; g++) {
#ifdef PADDLE_WITH_HIP
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::miopenConvolutionBackwardData(
                handle, &alpha, args.odesc.desc(),
                input_data + input_offset * g, args.wdesc.desc(),
                filter_data + filter_offset * g, args.cdesc.desc(), algo, &beta,
                args.idesc.desc(), transformed_output_data + output_offset * g,
                cudnn_workspace, workspace_size));
      };
#else   // PADDLE_WITH_HIP
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::hipdnnConvolutionBackwardData(
                handle, &alpha, args.wdesc.desc(),
                filter_data + filter_offset * g, args.odesc.desc(),
                input_data + input_offset * g, args.cdesc.desc(), algo,
                cudnn_workspace, workspace_size, &beta, args.idesc.desc(),
                transformed_output_data + output_offset * g));
      };
#endif  // PADDLE_WITH_HIP
      workspace_handle.RunFunc(cudnn_func, workspace_size);
    }
    if (!is_sys_pad && strides.size() == 2U) {
      Slice<paddle::platform::CUDADeviceContext, T, 4>(
          ctx, &transformed_output, output, starts, ends, axes);
    } else if (!is_sys_pad && strides.size() == 3U) {
      Slice<paddle::platform::CUDADeviceContext, T, 5>(
          ctx, &transformed_output, output, starts, ends, axes);
    }

    if (data_layout == platform::DataLayout::kNHWC) {
      Tensor output_transpose;
      Tensor output_nchw;
      output_nchw.ShareDataWith(*output);
      output_nchw.Resize(framework::make_ddim(output_vec));
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 2, 3, 1};
        DataTranspose<T, 4>(ctx, &output_nchw, &output_transpose, axis);
        *output = output_transpose;
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 2, 3, 4, 1};
        DataTranspose<T, 5>(ctx, &output_nchw, &output_transpose, axis);
        *output = output_transpose;
      }
    }
  }
};

template <typename T>
class CUDNNConvTransposeGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        paddle::platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    auto input = ctx.Input<Tensor>("Input");
    auto filter = ctx.Input<Tensor>("Filter");
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));
    const T* filter_data = filter->data<T>();

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    // cudnn v5 does not support dilations
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    int user_workspace_size = ctx.Attr<int>("workspace_size_MB");
    const std::string data_layout_str = ctx.Attr<std::string>("data_format");
    const paddle::platform::DataLayout data_layout =
        (data_layout_str != "NHWC" ? platform::DataLayout::kNCHW
                                   : platform::DataLayout::kNHWC);

    // if channel_last, transpose to channel_first
    Tensor input_transpose;
    Tensor output_grad_transpose;
    std::vector<int> input_vec = framework::vectorize<int>(input->dims());
    std::vector<int> output_vec =
        framework::vectorize<int>(output_grad->dims());
    if (data_layout == platform::DataLayout::kNHWC) {
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 3, 1, 2};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output_grad->dims()[axis[i]];
        }
        DataTranspose<T, 4>(ctx, input, &input_transpose, axis);
        DataTranspose<T, 4>(ctx, output_grad, &output_grad_transpose, axis);
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 4, 1, 2, 3};
        for (size_t i = 0; i < axis.size(); ++i) {
          input_vec[i] = input->dims()[axis[i]];
          output_vec[i] = output_grad->dims()[axis[i]];
        }
        DataTranspose<T, 5>(ctx, input, &input_transpose, axis);
        DataTranspose<T, 5>(ctx, output_grad, &output_grad_transpose, axis);
      }
    } else {
      input_transpose = *input;
      output_grad_transpose = *output_grad;
    }

    // update padding and dilation
    auto in_dims = input_transpose.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims;
    in_data_dims = framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    std::vector<int> input_pad(input_transpose.dims().size() * 2, 0);
    Tensor transformed_output_grad;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_output_grad_shape_vec(data_dim + 2);
      new_output_grad_shape_vec[0] = output_grad_transpose.dims()[0];
      new_output_grad_shape_vec[1] = output_grad_transpose.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_output_grad_shape_vec[i + 2] =
            output_grad_transpose.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_output_grad_shape(
          framework::make_ddim(new_output_grad_shape_vec));
      transformed_output_grad.Resize(new_output_grad_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_output_grad =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_output_grad_shape, dev_ctx);
      const int rank = input_transpose.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, output_grad_transpose, pad_value,
              &transformed_output_grad);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, output_grad_transpose, pad_value,
              &transformed_output_grad);
        } break;
        default:
          PADDLE_THROW(platform::errors::InvalidArgument(
              "Op(ConvTranspose) only supports 4-D or 5-D input Tensor."));
      }
    } else {
      transformed_output_grad = output_grad_transpose;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = input_transpose.data<T>();
    const T* output_grad_data = transformed_output_grad.data<T>();
    output_vec = framework::vectorize<int>(transformed_output_grad.dims());

    // ------------------- cudnn descriptors ---------------------
    platform::DataLayout layout;

    if (strides.size() == 2U) {
      layout = platform::DataLayout::kNCHW;
    } else {
      layout = platform::DataLayout::kNCDHW;
    }

    int iwo_groups = groups;
    int c_groups = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_groups = 1;
    c_groups = groups;
    groups = 1;
#endif

    auto dtype = platform::CudnnDataType<T>::type;

    ConvArgs args1{&transformed_output_grad,
                   filter,
                   &input_transpose,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args2{&transformed_output_grad,
                   filter,
                   &input_transpose,
                   strides,
                   padding_common,
                   dilations,
                   dtype};

#ifdef PADDLE_WITH_HIP
    miopenConvFwdAlgorithm_t data_algo{};
    miopenConvBwdWeightsAlgorithm_t filter_algo{};
#else
    hipdnnConvolutionFwdAlgo_t data_algo{};
    hipdnnConvolutionBwdFilterAlgo_t filter_algo{};
#endif

    auto layout_tensor = GetCudnnTensorFormat(layout);
    size_t workspace_size = 0;
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    bool deterministic = FLAGS_cudnn_deterministic;
    T* input_grad_data = nullptr;
    T* filter_grad_data = nullptr;
    if (input_grad)
      input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
    if (filter_grad)
      filter_grad_data = filter_grad->mutable_data<T>(ctx.GetPlace());

    if (input_grad) {
      input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
      args1.handle = handle;
      args1.idesc.set(transformed_output_grad, iwo_groups);
      args1.wdesc.set(*filter, layout_tensor, iwo_groups);
      args1.odesc.set(input_transpose, iwo_groups);
      args1.cdesc.set(dtype, padding_common, strides, dilations,
                      platform::AllowTF32Cudnn(), c_groups);
#ifdef PADDLE_WITH_HIP
      using search1 = SearchAlgorithm<miopenConvFwdAlgorithm_t>;
#else
      using search1 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
#endif
      data_algo = search1::Find<T>(args1, false, deterministic, ctx);
      workspace_size =
          std::max(workspace_size, search1::GetWorkspaceSize(args1, data_algo));
    }

    if (filter_grad) {
      filter_grad_data = filter_grad->mutable_data<T>(ctx.GetPlace());
      args2.handle = handle;
      args2.idesc.set(transformed_output_grad, iwo_groups);
      args2.wdesc.set(*filter_grad, layout_tensor, iwo_groups);
      args2.odesc.set(input_transpose, iwo_groups);
      args2.cdesc.set(dtype, padding_common, strides, dilations,
                      platform::AllowTF32Cudnn(), c_groups);
#ifdef PADDLE_WITH_HIP
      using search2 = SearchAlgorithm<miopenConvBwdWeightsAlgorithm_t>;
#else
      using search2 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
#endif
      filter_algo = search2::Find<T>(args2, false, deterministic, ctx);
      workspace_size = std::max(workspace_size,
                                search2::GetWorkspaceSize(args2, filter_algo));
    }

    // ------------------- cudnn conv backward data ---------------------
    // FIXME(typhoonzero): template type T may not be the same as cudnn call.
    int input_offset = input->numel() / input->dims()[0] / groups;
    int output_grad_offset = transformed_output_grad.numel() /
                             transformed_output_grad.dims()[0] / groups;
    int filter_offset = filter->numel() / groups;
    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = 0.0f;
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();
    if (input_grad) {
      // Because beta is zero, it is unnecessary to reset input_grad.
      for (int g = 0; g < groups; g++) {
#ifdef PADDLE_WITH_HIP
        auto cudnn_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::miopenConvolutionForward(
                  handle, &alpha, args1.idesc.desc(),
                  output_grad_data + output_grad_offset * g, args1.wdesc.desc(),
                  filter_data + filter_offset * g, args1.cdesc.desc(),
                  data_algo, &beta, args1.odesc.desc(),
                  input_grad_data + input_offset * g, cudnn_workspace,
                  workspace_size));
        };
#else   // PADDLE_WITH_HIP
        auto cudnn_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnConvolutionForward(
                  handle, &alpha, args1.idesc.desc(),
                  output_grad_data + output_grad_offset * g, args1.wdesc.desc(),
                  filter_data + filter_offset * g, args1.cdesc.desc(),
                  data_algo, cudnn_workspace, workspace_size, &beta,
                  args1.odesc.desc(), input_grad_data + input_offset * g));
        };
#endif  // PADDLE_WITH_HIP
        workspace_handle.RunFunc(cudnn_func, workspace_size);
      }

      if (data_layout == platform::DataLayout::kNHWC) {
        Tensor input_grad_transpose;
        Tensor input_grad_nchw;
        input_grad_nchw.ShareDataWith(*input_grad);
        input_grad_nchw.Resize(framework::make_ddim(input_vec));
        if (strides.size() == 2U) {
          std::vector<int> axis = {0, 2, 3, 1};
          DataTranspose<T, 4>(ctx, &input_grad_nchw, &input_grad_transpose,
                              axis);
          *input_grad = input_grad_transpose;
        } else if (strides.size() == 3U) {
          std::vector<int> axis = {0, 2, 3, 4, 1};
          DataTranspose<T, 5>(ctx, &input_grad_nchw, &input_grad_transpose,
                              axis);
          *input_grad = input_grad_transpose;
        }
      }
    }

    // ------------------- cudnn conv backward filter ---------------------
    if (filter_grad) {
      // Because beta is zero, it is unnecessary to reset filter_grad.
      // Gradient with respect to the filter
      for (int g = 0; g < groups; g++) {
#ifdef PADDLE_WITH_HIP
        auto cudnn_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::miopenConvolutionBackwardWeights(
                  handle, &alpha, args2.odesc.desc(),
                  input_data + input_offset * g, args2.idesc.desc(),
                  output_grad_data + output_grad_offset * g, args2.cdesc.desc(),
                  filter_algo, &beta, args2.wdesc.desc(),
                  filter_grad_data + filter_offset * g, cudnn_workspace,
                  workspace_size));
        };
#else   // PADDLE_WITH_HIP
        auto cudnn_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnConvolutionBackwardFilter(
                  handle, &alpha, args2.idesc.desc(),
                  output_grad_data + output_grad_offset * g, args2.odesc.desc(),
                  input_data + input_offset * g, args2.cdesc.desc(),
                  filter_algo, cudnn_workspace, workspace_size, &beta,
                  args2.wdesc.desc(), filter_grad_data + filter_offset * g));
        };
#endif  // PADDLE_WITH_HIP
        workspace_handle.RunFunc(cudnn_func, workspace_size);
      }
    }
  }
};

/*
 * Inputs:  I, W, dO, ddI, ddW
 * Outputs: ddO, dW, dI
 * ddo = conv_bp_data(W, ddI) + conv_bp_data(ddW, I)
 * dW = conv_bp_filter(dO, ddI)
 * dI = conv(dO, ddW)
 */
template <typename T>
class CUDNNConvTransposeDoubleGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        paddle::platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    auto X = ctx.Input<Tensor>("Input");
    auto W = ctx.Input<Tensor>("Filter");
    auto dO = ctx.Input<Tensor>("DOutput");
    auto ddX = ctx.Input<Tensor>("DDInput");
    auto ddW = ctx.Input<Tensor>("DDFilter");

    auto ddO = ctx.Output<Tensor>("DDOutput");
    auto dW = ctx.Output<Tensor>("DFilter");
    auto dX = ctx.Output<Tensor>("DInput");

    if (ddO) {
      ddO->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<platform::CUDADeviceContext, T> set_zero;
      set_zero(dev_ctx, ddO, static_cast<T>(0));
    }
    if (dW) {
      dW->mutable_data<T>(ctx.GetPlace());
    }
    if (dX) {
      dX->mutable_data<T>(ctx.GetPlace());
    }

    const T* dy = dO->data<T>();
    const T* w = W->data<T>();

    const T* ddx = nullptr;
    const T* ddw = nullptr;
    T *dw, *dx, *ddy;
    dw = dx = ddy = nullptr;
    T* transformed_dx = nullptr;
    const std::vector<int>& strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");

    bool deterministic = FLAGS_cudnn_deterministic;

    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");

    std::string padding_algorithm = ctx.Attr<std::string>("padding_algorithm");
    const std::string data_format = ctx.Attr<std::string>("data_format");
    const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

    // transform Tensors to channel first-----------
    Tensor transformed_X_channel(X->type());
    Tensor transformed_dO_channel(dO->type());
    Tensor transformed_ddX_channel(X->type());

    Tensor transformed_ddO_channel(dO->type());
    Tensor transformed_dX_channel(X->type());

    if (channel_last) {
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, X, &transformed_X_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, X, &transformed_X_channel);

      ResizeToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, dO, &transformed_dO_channel);
      TransToChannelFirst<platform::CUDADeviceContext, T>(
          ctx, dO, &transformed_dO_channel);

      if (ddX) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddX, &transformed_ddX_channel);
        TransToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddX, &transformed_ddX_channel);
      }

      if (ddO) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, ddO, &transformed_ddO_channel);
      }
      if (dX) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(
            ctx, dX, &transformed_dX_channel);
        transformed_dX_channel.mutable_data<T>(ctx.GetPlace());
      }

    } else {
      transformed_X_channel = *X;
      transformed_dO_channel = *dO;
      if (ddX) {
        transformed_ddX_channel = *ddX;
      }
      if (dX) {
        transformed_dX_channel = *dX;
      }
    }
    std::vector<int> output_vec =
        framework::vectorize<int>(transformed_dO_channel.dims());

    auto in_dims = transformed_X_channel.dims();
    auto filter_dims = W->dims();
    framework::DDim in_data_dims =
        framework::slice_ddim(in_dims, 2, in_dims.size());
    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);
    Tensor transformed_X(X->type());
    Tensor transformed_ddX(X->type());

    Tensor transformed_dO(dO->type());

    std::vector<int> padding_common(data_dim, 0);
    std::vector<int> input_pad(X->dims().size() * 2, 0);

    if (!is_sys_pad) {
      // get pad
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      std::vector<int> new_output_grad_shape_vec(data_dim + 2);

      new_input_shape_vec[0] = transformed_X_channel.dims()[0];
      new_input_shape_vec[1] = transformed_X_channel.dims()[1];

      new_output_grad_shape_vec[0] = transformed_dO_channel.dims()[0];
      new_output_grad_shape_vec[1] = transformed_dO_channel.dims()[1];

      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            transformed_X_channel.dims()[i + 2] + padding_diff[i];

        new_output_grad_shape_vec[i + 2] =
            transformed_dO_channel.dims()[i + 2] + padding_diff[i];

        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_X.Resize(new_input_shape);
      transformed_ddX.Resize(new_input_shape);

      framework::DDim new_output_grad_shape(
          framework::make_ddim(new_output_grad_shape_vec));
      transformed_dO.Resize(new_output_grad_shape);

      transformed_dO =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_output_grad_shape, dev_ctx);

      transformed_X =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      if (ddX) {
        transformed_ddX =
            ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
                new_input_shape, dev_ctx);
      }

      // pad for input
      const int rank = X->dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
          if (dO) {
            math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
                ctx, input_pad, transformed_dO_channel, pad_value,
                &transformed_dO);
          }

          if (ddX) {
            math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
                ctx, input_pad, transformed_ddX_channel, pad_value,
                &transformed_ddX);
          }
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
          if (ddX) {
            math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
                ctx, input_pad, transformed_ddX_channel, pad_value,
                &transformed_ddX);
          }
        } break;
        default:
          PADDLE_THROW(platform::errors::InvalidArgument(
              "ConvOp only support tensors with 4 or 5 dimensions."));
      }

    } else {
      transformed_X = transformed_X_channel;
      transformed_dO = transformed_dO_channel;
      if (ddX) {
        transformed_ddX = transformed_ddX_channel;
      }

      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    std::vector<int64_t> starts(data_dim, 0);
    std::vector<int64_t> ends(data_dim, 0);
    std::vector<int64_t> axes(data_dim, 0);
    for (size_t i = 0; i < data_dim; ++i) {
      starts[i] = input_pad[2 * i + 4] * (strides[i] + 1);
      ends[i] = starts[i] + output_vec[i + 2];
      axes[i] = i + 2;
    }

    std::vector<int> transformed_output_vec = output_vec;
    for (size_t i = 0; i < data_dim; ++i) {
      transformed_output_vec[i + 2] =
          output_vec[i + 2] +
          (input_pad[2 * i + 4] + input_pad[2 * i + 5]) * strides[i] -
          2 * padding_common[i] + paddings[2 * i] + paddings[2 * i + 1];
    }

    if (!is_sys_pad) {
      DDim transformed_output_shape(
          framework::make_ddim(transformed_output_vec));
      transformed_ddO_channel.mutable_data<T>(transformed_output_shape,
                                              ctx.GetPlace());
    } else {
      ddO->mutable_data<T>(ctx.GetPlace());
      transformed_ddO_channel = *ddO;
      transformed_ddO_channel.Resize(
          framework::make_ddim(transformed_output_vec));
    }

    const T* x = transformed_X.data<T>();

    int iwo_group = groups;
    int c_group = 1;
#if CUDNN_VERSION_MIN(7, 0, 1)
    iwo_group = 1;
    c_group = groups;
    groups = 1;
#endif
    auto dtype = platform::CudnnDataType<T>::type;

    auto handle = dev_ctx.cudnn_handle();

    ConvArgs args1{&transformed_ddO_channel,
                   W,
                   &transformed_ddX,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args2{&transformed_ddO_channel, ddW,       &transformed_X, strides,
                   padding_common,           dilations, dtype};

    ConvArgs args3{&transformed_dO,
                   dW,
                   &transformed_ddX_channel,
                   strides,
                   padding_common,
                   dilations,
                   dtype};
    ConvArgs args4{
        &transformed_dO, ddW,  &transformed_dX_channel, strides, padding_common,
        dilations,       dtype};
#ifdef PADDLE_WITH_HIP
    miopenConvBwdDataAlgorithm_t bwd_algo1 =
        static_cast<miopenConvBwdDataAlgorithm_t>(0);
    miopenConvBwdDataAlgorithm_t bwd_algo2 =
        static_cast<miopenConvBwdDataAlgorithm_t>(0);
    miopenConvFwdAlgorithm_t data_algo =
        static_cast<miopenConvFwdAlgorithm_t>(0);
    miopenConvBwdWeightsAlgorithm_t filter_algo =
        static_cast<miopenConvBwdWeightsAlgorithm_t>(0);
#else
    hipdnnConvolutionBwdDataAlgo_t bwd_algo1 =
        static_cast<hipdnnConvolutionBwdDataAlgo_t>(0);
    hipdnnConvolutionBwdDataAlgo_t bwd_algo2 =
        static_cast<hipdnnConvolutionBwdDataAlgo_t>(0);
    hipdnnConvolutionFwdAlgo_t data_algo =
        static_cast<hipdnnConvolutionFwdAlgo_t>(0);
    hipdnnConvolutionBwdFilterAlgo_t filter_algo =
        static_cast<hipdnnConvolutionBwdFilterAlgo_t>(0);
#endif

    auto layout = GetCudnnTensorFormat(platform::DataLayout::kNCHW);

    // ddo = conv(ddI, W) + conv(I, ddW)
    size_t workspace_size = 0;

    T* transformed_ddy_channel = nullptr;

    if (ddO) {
      ddy = ddO->data<T>();
      transformed_ddy_channel = transformed_ddO_channel.data<T>();
      if (ddX) {
        args1.handle = handle;
        args1.idesc.set(transformed_ddO_channel, iwo_group);
        args1.wdesc.set(*W, layout, iwo_group);
        args1.odesc.set(transformed_ddX, iwo_group);
        args1.cdesc.set(dtype, padding_common, strides, dilations, c_group);
#ifdef PADDLE_WITH_HIP
        using search1 = SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
#else
        using search1 = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
#endif
        bwd_algo1 = search1::Find<T>(args1, false, deterministic, ctx);
        workspace_size = search1::GetWorkspaceSize(args1, bwd_algo1);
      }

      if (ddW) {
        ddw = ddW->data<T>();
        args2.handle = handle;
        args2.idesc.set(transformed_ddO_channel, iwo_group);
        args2.wdesc.set(*ddW, layout, iwo_group);
        args2.odesc.set(transformed_X, iwo_group);
        args2.cdesc.set(dtype, padding_common, strides, dilations, c_group);
#ifdef PADDLE_WITH_HIP
        using search2 = SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
#else
        using search2 = SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
#endif
        bwd_algo2 = search2::Find<T>(args2, false, deterministic, ctx);
        workspace_size = std::max(workspace_size,
                                  search2::GetWorkspaceSize(args2, bwd_algo2));
      }
    }

    if (dW && ddX) {
      dw = dW->data<T>();
      args3.handle = handle;
      args3.idesc.set(transformed_dO, iwo_group);
      args3.wdesc.set(*dW, layout, iwo_group);

      args3.odesc.set(transformed_ddX_channel, iwo_group);

      args3.cdesc.set(dtype, padding_common, strides, dilations, c_group);
#ifdef PADDLE_WITH_HIP
      using search3 = SearchAlgorithm<miopenConvBwdWeightsAlgorithm_t>;
#else
      using search3 = SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
#endif
      filter_algo = search3::Find<T>(args3, false, deterministic, ctx);
      workspace_size = std::max(workspace_size,
                                search3::GetWorkspaceSize(args3, filter_algo));
    }

    if (ddW && dX) {
      transformed_dx = transformed_dX_channel.data<T>();

      args4.handle = handle;
      args4.idesc.set(transformed_dO, iwo_group);
      args4.wdesc.set(*ddW, layout, iwo_group);
      args4.odesc.set(transformed_dX_channel, iwo_group);
      args4.cdesc.set(dtype, padding_common, strides, dilations, c_group);
#ifdef PADDLE_WITH_HIP
      using search4 = SearchAlgorithm<miopenConvFwdAlgorithm_t>;
#else
      using search4 = SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
#endif
      data_algo = search4::Find<T>(args4, false, deterministic, ctx);
      workspace_size =
          std::max(workspace_size, search4::GetWorkspaceSize(args4, data_algo));
    }

    int i_n, i_c, i_d, i_h, i_w;
    GetNCDHW(transformed_X.dims(), platform::DataLayout::kNCHW, &i_n, &i_c,
             &i_d, &i_h, &i_w);

    int o_n, o_c, o_d, o_h, o_w;
    GetNCDHW(transformed_dO.dims(), platform::DataLayout::kNCHW, &o_n, &o_c,
             &o_d, &o_h, &o_w);

    int group_offset_in =
        transformed_X.numel() / transformed_X.dims()[0] / groups;
    int group_offset_out =
        transformed_dO.numel() / transformed_dO.dims()[0] / groups;
    int group_offset_filter = W->numel() / groups;

    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = 0.0f;

    auto wkspace_handle = dev_ctx.cudnn_workspace_handle();

    if (ddO) {
      if (ddX) {
        ddx = transformed_ddX.data<T>();
        for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                PADDLE_ENFORCE_CUDA_SUCCESS(
                    platform::dynload::miopenConvolutionBackwardData(
                        handle, &alpha, args1.odesc.desc(),
                        ddx + i * group_offset_in, args1.wdesc.desc(),
                        w + i * group_offset_filter, args1.cdesc.desc(),
                        bwd_algo1, &beta, args1.idesc.desc(),
                        transformed_ddy_channel + i * group_offset_out,
                        workspace_ptr, workspace_size));
              },
              workspace_size);
#else   // PADDLE_WITH_HIP
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                PADDLE_ENFORCE_CUDA_SUCCESS(
                    platform::dynload::hipdnnConvolutionBackwardData(
                        handle, &alpha, args1.wdesc.desc(),
                        w + i * group_offset_filter, args1.odesc.desc(),
                        ddx + i * group_offset_in, args1.cdesc.desc(),
                        bwd_algo1, workspace_ptr, workspace_size, &beta,
                        args1.idesc.desc(),
                        transformed_ddy_channel + i * group_offset_out));
              },
              workspace_size);
#endif  // PADDLE_WITH_HIP
        }
      }
      if (ddW) {
        for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                PADDLE_ENFORCE_CUDA_SUCCESS(
                    platform::dynload::miopenConvolutionBackwardData(
                        handle, &alpha, args2.odesc.desc(),
                        x + i * group_offset_in, args2.wdesc.desc(),
                        ddw + i * group_offset_filter, args2.cdesc.desc(),
                        bwd_algo2, &alpha, args2.idesc.desc(),
                        transformed_ddy_channel + i * group_offset_out,
                        workspace_ptr, workspace_size));
              },
              workspace_size);
#else   // PADDLE_WITH_HIP
          wkspace_handle.RunFunc(
              [&](void* workspace_ptr) {
                PADDLE_ENFORCE_CUDA_SUCCESS(
                    platform::dynload::hipdnnConvolutionBackwardData(
                        handle, &alpha, args2.wdesc.desc(),
                        ddw + i * group_offset_filter, args2.odesc.desc(),
                        x + i * group_offset_in, args2.cdesc.desc(), bwd_algo2,
                        workspace_ptr, workspace_size, &alpha,
                        args2.idesc.desc(),
                        transformed_ddy_channel + i * group_offset_out));
              },
              workspace_size);
#endif  // PADDLE_WITH_HIP
        }
      }
      if ((!is_sys_pad) && (!channel_last)) {
        if (strides.size() == 2U) {
          Slice<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, &transformed_ddO_channel, ddO, starts, ends, axes);
        } else if (!is_sys_pad && strides.size() == 3U) {
          Slice<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, &transformed_ddO_channel, ddO, starts, ends, axes);
        }
      } else if ((!is_sys_pad) && (channel_last)) {
        if (strides.size() == 2U) {
          Slice<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, &transformed_ddO_channel, &transformed_ddO_channel, starts,
              ends, axes);
        } else if (!is_sys_pad && strides.size() == 3U) {
          Slice<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, &transformed_ddO_channel, &transformed_ddO_channel, starts,
              ends, axes);
        }

        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_ddO_channel, ddO);
      }
    }

    T* transformed_dy_channel = transformed_dO.data<T>();
    if (dW && ddX) {
      ddx = transformed_ddX_channel.data<T>();
      for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::miopenConvolutionBackwardWeights(
                      handle, &alpha, args3.odesc.desc(),
                      ddx + i * group_offset_in, args3.idesc.desc(),
                      transformed_dy_channel + i * group_offset_out,
                      args3.cdesc.desc(), filter_algo, &beta,
                      args3.wdesc.desc(), dw + i * group_offset_filter,
                      workspace_ptr, workspace_size));
            },
            workspace_size);
#else   // PADDLE_WITH_HIP
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::hipdnnConvolutionBackwardFilter(
                      handle, &alpha, args3.idesc.desc(),
                      transformed_dy_channel + i * group_offset_out,
                      args3.odesc.desc(), ddx + i * group_offset_in,
                      args3.cdesc.desc(), filter_algo, workspace_ptr,
                      workspace_size, &beta, args3.wdesc.desc(),
                      dw + i * group_offset_filter));
            },
            workspace_size);
#endif  // PADDLE_WITH_HIP
      }
    }

    if (dX && ddW) {
      ddw = ddW->data<T>();
      for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::miopenConvolutionForward(
                      handle, &alpha, args4.idesc.desc(),
                      transformed_dy_channel + i * group_offset_out,
                      args4.wdesc.desc(), ddw + i * group_offset_filter,
                      args4.cdesc.desc(), data_algo, &beta, args4.odesc.desc(),
                      transformed_dx + i * group_offset_in, workspace_ptr,
                      workspace_size));
            },
            workspace_size);
#else   // PADDLE_WITH_HIP
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_CUDA_SUCCESS(
                  platform::dynload::hipdnnConvolutionForward(
                      handle, &alpha, args4.idesc.desc(),
                      transformed_dy_channel + i * group_offset_out,
                      args4.wdesc.desc(), ddw + i * group_offset_filter,
                      args4.cdesc.desc(), data_algo, workspace_ptr,
                      workspace_size, &beta, args4.odesc.desc(),
                      transformed_dx + i * group_offset_in));
            },
            workspace_size);
#endif  // PADDLE_WITH_HIP
      }
      if (channel_last) {
        TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
            ctx, &transformed_dX_channel, dX);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_KERNEL(conv2d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeOpKernel<float>);
REGISTER_OP_KERNEL(conv2d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeGradOpKernel<float>);
REGISTER_OP_KERNEL(
    conv2d_transpose_grad_grad, CUDNN, plat::CUDAPlace,
    paddle::operators::CUDNNConvTransposeDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvTransposeDoubleGradOpKernel<plat::float16>);

REGISTER_OP_KERNEL(conv3d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeOpKernel<float>);
REGISTER_OP_KERNEL(conv3d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeGradOpKernel<float>);
#else
REGISTER_OP_KERNEL(conv2d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeOpKernel<float>,
                   ops::CUDNNConvTransposeOpKernel<double>);
REGISTER_OP_KERNEL(conv2d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeGradOpKernel<float>,
                   ops::CUDNNConvTransposeGradOpKernel<double>);
REGISTER_OP_KERNEL(
    conv2d_transpose_grad_grad, CUDNN, plat::CUDAPlace,
    paddle::operators::CUDNNConvTransposeDoubleGradOpKernel<float>,
    paddle::operators::CUDNNConvTransposeDoubleGradOpKernel<double>,
    paddle::operators::CUDNNConvTransposeDoubleGradOpKernel<plat::float16>);

REGISTER_OP_KERNEL(conv3d_transpose, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeOpKernel<float>,
                   ops::CUDNNConvTransposeOpKernel<double>);
REGISTER_OP_KERNEL(conv3d_transpose_grad, CUDNN, ::paddle::platform::CUDAPlace,
                   ops::CUDNNConvTransposeGradOpKernel<plat::float16>,
                   ops::CUDNNConvTransposeGradOpKernel<float>,
                   ops::CUDNNConvTransposeGradOpKernel<double>);
#endif
