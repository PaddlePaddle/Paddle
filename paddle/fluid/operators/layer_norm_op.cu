#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hipcub/hipcub.hpp>
#include "paddle/fluid/operators/layer_norm_op.h"

namespace paddle {
namespace operators {

inline static int GetDesiredBlockDim(int block_dim) {
  const int kMaxBlockDim = 512;
  return block_dim >= kMaxBlockDim
             ? kMaxBlockDim
             : (1 << (static_cast<int>(std::log2f(block_dim))));
}

#define FIXED_BLOCK_DIM_CASE_BASE(log2_block_dim, ...)  \
  case (1 << (log2_block_dim)): {                       \
    constexpr auto kBlockDim = (1 << (log2_block_dim)); \
    __VA_ARGS__;                                        \
  } break

#define FIXED_BLOCK_DIM_CASE(...)              \
  FIXED_BLOCK_DIM_CASE_BASE(9, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(8, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(7, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(6, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(5, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(4, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(3, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(2, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(1, ##__VA_ARGS__)

static __device__ __forceinline__ float real_sqrt(float x) { return sqrtf(x); }
static __device__ __forceinline__ double real_sqrt(double x) { return sqrt(x); }

template <typename T>
struct PairForLayerNorm {
  __device__ __forceinline__ PairForLayerNorm() {}
  __device__ __forceinline__ PairForLayerNorm(const T &first, const T &second)
      : first_(first), second_(second) {}

  T first_;
  T second_;
};

template <typename T>
struct PairForLayerNormAddFunctor {
  __device__ __forceinline__ PairForLayerNorm<T> operator()(
      const PairForLayerNorm<T> &p1, const PairForLayerNorm<T> &p2) {
    return PairForLayerNorm<T>(p1.first_ + p2.first_, p1.second_ + p2.second_);
  }
};

template <typename T, int BlockDim>
__global__ void LayerNormForward(const T *x, const T *scale, const T *bias,
                                 T *y, T *mean, T *var, float epsilon,
                                 int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<double>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  // Step 1: Reduce to calculate mean and var
  double mean_val = 0;
  double var_val = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T tmp = x[i];
    mean_val += tmp;
    var_val += (tmp * tmp);
  }
  auto pair = BlockReduce(temp_storage)
                  .Reduce(PairForLayerNorm<double>(mean_val, var_val),
                          PairForLayerNormAddFunctor<double>());
  if (threadIdx.x == 0) {
    auto tmp = pair.first_ / feature_size;
    mean[blockIdx.x] = static_cast<T>(tmp);
    var[blockIdx.x] = static_cast<T>(pair.second_ / feature_size - tmp * tmp);
  }
  __syncthreads();
  mean_val = mean[blockIdx.x];
  var_val = static_cast<T>(real_sqrt(var[blockIdx.x] + epsilon));

  // Step 2: Calculate y
  if (scale != nullptr) {
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = scale[j] * (x[i] - mean_val) / var_val + bias[j];
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = scale[j] * (x[i] - mean_val) / var_val;
      }
    }
  } else {  // scale == nullptr
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = (x[i] - mean_val) / var_val + bias[j];
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = (x[i] - mean_val) / var_val;
      }
    }
  }
}

// Make sure that d_scale != nullptr && d_bias != nullptr
// Since d_scale != nullptr, scale would not be nullptr
template <typename T, int BlockDim, bool HasDx>
__global__ void LayerNormBackwardGradientAll(const T *x, const T *d_y,
                                             T *d_scale, T *d_bias, T *d_x,
                                             const T *mean, const T *var,
                                             const T *scale, float epsilon,
                                             int batch_size, int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<T>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = threadIdx.x * feature_size + blockIdx.x;
  int end_idx = batch_size * feature_size + blockIdx.x;
  int stride = BlockDim * feature_size;

  T d_scale_partial = 0, d_bias_partial = 0;

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val = static_cast<T>(real_sqrt(var[row_idx] + epsilon));
    d_scale_partial += d_y[i] * (x[i] - mean[row_idx]) / var_val;
    d_bias_partial += d_y[i];
    if (HasDx) {
      d_x[i] = d_y[i] * scale[blockIdx.x] / var_val;
    }
  }

  auto pair = BlockReduce(temp_storage)
                  .Reduce(PairForLayerNorm<T>(d_scale_partial, d_bias_partial),
                          PairForLayerNormAddFunctor<T>());

  if (threadIdx.x == 0) {
    d_scale[blockIdx.x] = pair.first_;
    d_bias[blockIdx.x] = pair.second_;
  }
}

// Make sure that there is only one true expression: d_scale != nullptr
// or d_bias != nullptr
// Notice: scale may be nullptr
template <typename T, int BlockDim, bool HasDx, bool HasDScale>
__global__ void LayerNormBackwardGradientScaleOrBias(
    const T *x, const T *d_y, T *d_scale, T *d_bias, T *d_x, const T *mean,
    const T *var, const T *scale, float epsilon, int batch_size,
    int feature_size) {
  using BlockReduce = hipcub::BlockReduce<T, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int beg_idx = threadIdx.x * feature_size + blockIdx.x;
  int end_idx = batch_size * feature_size + blockIdx.x;
  int stride = BlockDim * feature_size;
  T d_scale_or_d_bias_partial = 0;

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val = static_cast<T>(real_sqrt(var[row_idx] + epsilon));
    if (HasDScale) {
      d_scale_or_d_bias_partial += d_y[i] * (x[i] - mean[row_idx]) / var_val;
    } else {  // d_bias != nullptr
      d_scale_or_d_bias_partial += d_y[i];
    }

    if (HasDx) {
      if (scale != nullptr) {
        d_x[i] = d_y[i] * scale[blockIdx.x] / var_val;
      } else {
        d_x[i] = d_y[i] / var_val;
      }
    }
  }

  d_scale_or_d_bias_partial =
      BlockReduce(temp_storage).Reduce(d_scale_or_d_bias_partial, hipcub::Sum());

  if (threadIdx.x == 0) {
    if (HasDScale) {
      d_scale[blockIdx.x] = d_scale_or_d_bias_partial;
    } else {
      d_bias[blockIdx.x] = d_scale_or_d_bias_partial;
    }
  }
}

template <typename T, int BlockDim>
__global__ void LayerNormBackwardPostProcessToCalculateDX(const T *x, T *d_x,
                                                          const T *mean,
                                                          const T *var,
                                                          float epsilon,
                                                          int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<T>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T d_x_reduce_tmp[2];

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  T block_mean = mean[blockIdx.x];
  T block_var = var[blockIdx.x];
  T d_x_mean_partial = 0, d_x_var_partial = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x_mean_partial += d_x[i];
    d_x_var_partial += d_x[i] * (x[i] - block_mean);
  }

  auto pair =
      BlockReduce(temp_storage)
          .Reduce(PairForLayerNorm<T>(d_x_mean_partial, d_x_var_partial),
                  PairForLayerNormAddFunctor<T>());

  if (threadIdx.x == 0) {
    d_x_reduce_tmp[0] = pair.first_ / feature_size;
    d_x_reduce_tmp[1] = pair.second_ / (feature_size * (block_var + epsilon));
  }
  __syncthreads();

  d_x_mean_partial = d_x_reduce_tmp[0];
  d_x_var_partial = d_x_reduce_tmp[1];
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x[i] -= d_x_mean_partial;
    d_x[i] -= (x[i] - block_mean) * d_x_var_partial;
  }
}

// Here, we only calculate d_x
template <typename T, int BlockDim>
__global__ void LayerNormBackwardGradientOnlyDX(const T *x, const T *d_y,
                                                T *d_x, const T *mean,
                                                const T *var, const T *scale,
                                                float epsilon,
                                                int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<T>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T d_x_reduce_tmp[2];

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  T block_mean = mean[blockIdx.x], block_var = var[blockIdx.x];
  T d_x_mean_partial = 0, d_x_var_partial = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    auto var_val = static_cast<T>(real_sqrt(block_var + epsilon));
    if (scale != nullptr) {
      int col_idx = i % feature_size;
      d_x[i] = d_y[i] * scale[col_idx] / var_val;
    } else {
      d_x[i] = d_y[i] / var_val;
    }
    d_x_mean_partial += d_x[i];
    d_x_var_partial += d_x[i] * (x[i] - block_mean);
  }

  auto pair =
      BlockReduce(temp_storage)
          .Reduce(PairForLayerNorm<T>(d_x_mean_partial, d_x_var_partial),
                  PairForLayerNormAddFunctor<T>());

  if (threadIdx.x == 0) {
    d_x_reduce_tmp[0] = pair.first_ / feature_size;
    d_x_reduce_tmp[1] = pair.second_ / (feature_size * (block_var + epsilon));
  }
  __syncthreads();

  d_x_mean_partial = d_x_reduce_tmp[0];
  d_x_var_partial = d_x_reduce_tmp[1];
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x[i] -= d_x_mean_partial;
    d_x[i] -= (x[i] - block_mean) * d_x_var_partial;
  }
}

template <typename T>
__global__ void LayerNormBackwardWhenBatchSizeIsOne(
    const T *x, const T *d_y, T *d_x, T *d_scale, T *d_bias, const T *mean,
    const T *var, const T *scale, float epsilon, int feature_size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < feature_size) {
    auto var_val = static_cast<T>(real_sqrt(var[idx] + epsilon));
    if (d_x != nullptr) {
      if (d_scale == nullptr) {
        d_x[idx] = d_y[idx] / var_val;
      } else {
        d_x[idx] = d_y[idx] * scale[idx] / var_val;
      }
    }

    if (d_scale != nullptr) {
      d_scale[idx] = d_y[idx] * (x[idx] - mean[idx]) / var_val;
    }

    if (d_bias != nullptr) d_bias[idx] = d_y[idx];
  }
}

template <typename T>
static void LayerNormBackward(const T *x, const T *d_y, const T *scale,
                              const T *mean, const T *var, T *d_x, T *d_scale,
                              T *d_bias, float epsilon, int batch_size,
                              int feature_size, hipStream_t stream) {
  const int kMaxBlockDim = 512;
  int gradient_flag = ((d_x != nullptr ? 1 : 0) << 2) |
                      ((d_scale != nullptr ? 1 : 0) << 1) |
                      ((d_bias != nullptr ? 1 : 0));
  if (gradient_flag == 0) return;

  if (batch_size == 1) {
    LayerNormBackwardWhenBatchSizeIsOne<
        T><<<(feature_size + kMaxBlockDim - 1) / kMaxBlockDim, kMaxBlockDim, 0,
             stream>>>(x, d_y, d_x, d_scale, d_bias, mean, var, scale, epsilon,
                       feature_size);

    if (d_x != nullptr) {
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardPostProcessToCalculateDX<
                             T, kBlockDim><<<1, kBlockDim, 0, stream>>>(
            x, d_x, mean, var, epsilon, feature_size));
      }
    }
    return;
  }

  auto block_dim = GetDesiredBlockDim(batch_size);
  switch (gradient_flag) {
    case 1:  // d_x == nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, false,
                             false><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      break;
    case 2:  // d_x == nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, false,
                             true><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      break;
    case 3:  // d_x == nullptr, d_scale != nulptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientAll<
                T, kBlockDim, false><<<feature_size, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size));
      }
      break;
    case 4:  // d_x != nullptr, d_scale == nullptr, d_bias == nullptr
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientOnlyDX<
                T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_y, d_x, mean, var, scale, epsilon, feature_size));
      }
      break;
    case 5:  // d_x != nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, true,
                             false><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    case 6:  // d_x != nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, true,
                             true><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    case 7:  // d_x != nullptr, d_scale != nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientAll<
                T, kBlockDim, true><<<feature_size, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    default:
      break;
  }
}

template <typename T>
class LayerNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *x = ctx.Input<Tensor>("X");

    auto *y = ctx.Output<Tensor>("Y");
    auto *mean = ctx.Output<Tensor>("Mean");
    auto *var = ctx.Output<Tensor>("Variance");
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");

    const auto x_dims = x->dims();
    auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    auto *mean_data = mean->mutable_data<T>(ctx.GetPlace());
    auto *var_data = var->mutable_data<T>(ctx.GetPlace());
    auto *scale_data = (scale == nullptr ? nullptr : scale->data<T>());
    auto *bias_data = (bias == nullptr ? nullptr : bias->data<T>());

    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    switch (GetDesiredBlockDim(feature_size)) {
      FIXED_BLOCK_DIM_CASE(
          LayerNormForward<T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
              x_data, scale_data, bias_data, y_data, mean_data, var_data,
              epsilon, feature_size));
      default:
        PADDLE_THROW(
            "Product from begin_norm_axis to end must be larger than 1");
        break;
    }
  }
};

template <typename T>
class LayerNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const float epsilon = ctx.Attr<float>("epsilon");
    // d_x, d_scale, d_bias may be nullptr
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *x = ctx.Input<Tensor>("X");
    auto *mean = ctx.Input<Tensor>("Mean");
    auto *var = ctx.Input<Tensor>("Variance");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    auto *x_data = x->data<T>();
    auto *d_y_data = d_y->data<T>();
    auto *mean_data = mean->data<T>();
    auto *var_data = var->data<T>();
    auto *scale_data = (scale == nullptr ? nullptr : scale->data<T>());
    auto *d_scale_data =
        (d_scale == nullptr ? nullptr
                            : d_scale->mutable_data<T>(ctx.GetPlace()));
    auto *d_bias_data =
        (d_bias == nullptr ? nullptr : d_bias->mutable_data<T>(ctx.GetPlace()));
    auto *d_x_data =
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace()));

    const auto &x_dims = x->dims();
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    LayerNormBackward<T>(x_data, d_y_data, scale_data, mean_data, var_data,
                         d_x_data, d_scale_data, d_bias_data, epsilon,
                         batch_size, feature_size, stream);
  }
};

#undef FIXED_BLOCK_DIM_CASE_BASE
#undef FIXED_BLOCK_DIM_CASE
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>);
