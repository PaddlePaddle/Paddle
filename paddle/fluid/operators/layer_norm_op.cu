#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hipcub/hipcub.hpp>
#include <memory>
#include <vector>

#include "paddle/fluid/framework/ddim.h"
#include "paddle/fluid/operators/layer_norm_op.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using LayerNormParamType = typename CudnnDataType<T>::BatchNormParamType;

inline static int GetDesiredBlockDim(int block_dim) {
  const int kMaxBlockDim = 512;
  return block_dim >= kMaxBlockDim
             ? kMaxBlockDim
             : (1 << (static_cast<int>(std::log2f(block_dim))));
}

#define FIXED_BLOCK_DIM_CASE_BASE(log2_block_dim, ...)  \
  case (1 << (log2_block_dim)): {                       \
    constexpr auto kBlockDim = (1 << (log2_block_dim)); \
    __VA_ARGS__;                                        \
  } break

#define FIXED_BLOCK_DIM_CASE(...)              \
  FIXED_BLOCK_DIM_CASE_BASE(9, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(8, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(7, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(6, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(5, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(4, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(3, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(2, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(1, ##__VA_ARGS__)

#define FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(                             \
    log2_block_dim, feature_size, kMaxBlockNum, ...)                           \
  case (1 << (log2_block_dim)): {                                              \
    for (int i = 0; i < std::ceil(feature_size / (1.0 * kMaxBlockNum)); i++) { \
      int col_offset = i * kMaxBlockNum;                                       \
      int block_num = std::min(feature_size - col_offset, kMaxBlockNum);       \
      constexpr auto kBlockDim = (1 << (log2_block_dim));                      \
      __VA_ARGS__;                                                             \
    }                                                                          \
  } break

#define FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(feature_size, kMaxBlockNum, ...) \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(9, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(8, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(7, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(6, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(5, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(4, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(3, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(2, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(1, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__)

static __device__ __forceinline__ float real_sqrt(float x) { return sqrtf(x); }
static __device__ __forceinline__ double real_sqrt(double x) { return sqrt(x); }

template <typename T>
struct PairForLayerNorm {
  __device__ __forceinline__ PairForLayerNorm() {}
  __device__ __forceinline__ PairForLayerNorm(const T &first, const T &second)
      : first_(first), second_(second) {}

  T first_;
  T second_;
};

template <typename T>
struct PairForLayerNormAddFunctor {
  __device__ __forceinline__ PairForLayerNorm<T> operator()(
      const PairForLayerNorm<T> &p1, const PairForLayerNorm<T> &p2) {
    return PairForLayerNorm<T>(p1.first_ + p2.first_, p1.second_ + p2.second_);
  }
};

template <typename T, typename U, int BlockDim>
__global__ void LayerNormForward(const T *x, const U *scale, const U *bias,
                                 T *y, U *mean, U *var, float epsilon,
                                 int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<double>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  // Step 1: Reduce to calculate mean and var
  double mean_val = 0;
  double var_val = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    U tmp = static_cast<U>(x[i]);
    mean_val += tmp;
    var_val += (tmp * tmp);
  }
  auto pair = BlockReduce(temp_storage)
                  .Reduce(PairForLayerNorm<double>(mean_val, var_val),
                          PairForLayerNormAddFunctor<double>());
  if (threadIdx.x == 0) {
    auto tmp = pair.first_ / feature_size;
    mean[blockIdx.x] = static_cast<U>(tmp);
    var[blockIdx.x] = static_cast<U>(pair.second_ / feature_size - tmp * tmp);
  }
  __syncthreads();
  mean_val = mean[blockIdx.x];
  var_val = static_cast<U>(real_sqrt(var[blockIdx.x]) + epsilon);

  // Step 2: Calculate y
  if (scale != nullptr) {
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>(
            scale[j] * (static_cast<U>(x[i]) - mean_val) / var_val + bias[j]);
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>(scale[j] * (static_cast<U>(x[i]) - mean_val) /
                              var_val);
      }
    }
  } else {  // scale == nullptr
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>((static_cast<U>(x[i]) - mean_val) / var_val +
                              bias[j]);
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>((static_cast<U>(x[i]) - mean_val) / var_val);
      }
    }
  }
}

// Make sure that d_scale != nullptr && d_bias != nullptr
// Since d_scale != nullptr, scale would not be nullptr
template <typename T, typename U, int BlockDim, bool HasDx>
__global__ void LayerNormBackwardGradientAll(const T *x, const T *d_y,
                                             U *d_scale, U *d_bias, T *d_x,
                                             const U *mean, const U *var,
                                             const U *scale, float epsilon,
                                             int batch_size, int feature_size,
                                             int col_offset) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = threadIdx.x * feature_size + (blockIdx.x + col_offset);
  int end_idx = batch_size * feature_size + (blockIdx.x + col_offset);
  int stride = BlockDim * feature_size;

  U d_scale_partial = static_cast<U>(0), d_bias_partial = static_cast<U>(0);

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val = real_sqrt(static_cast<U>(var[row_idx]) + epsilon);
    d_scale_partial += static_cast<U>(d_y[i]) *
                       (static_cast<U>(x[i]) - mean[row_idx]) / var_val;
    d_bias_partial += static_cast<U>(d_y[i]);
    if (HasDx) {
      d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) *
                              scale[blockIdx.x + col_offset] / var_val);
    }
  }

  auto pair = BlockReduce(temp_storage)
                  .Reduce(PairForLayerNorm<U>(d_scale_partial, d_bias_partial),
                          PairForLayerNormAddFunctor<U>());

  if (threadIdx.x == 0) {
    d_scale[blockIdx.x + col_offset] = pair.first_;
    d_bias[blockIdx.x + col_offset] = pair.second_;
  }
}

// Make sure that there is only one true expression: d_scale != nullptr
// or d_bias != nullptr
// Notice: scale may be nullptr
template <typename T, typename U, int BlockDim, bool HasDx, bool HasDScale>
__global__ void LayerNormBackwardGradientScaleOrBias(
    const T *x, const T *d_y, U *d_scale, U *d_bias, T *d_x, const U *mean,
    const U *var, const U *scale, float epsilon, int batch_size,
    int feature_size, int col_offset) {
  using BlockReduce = hipcub::BlockReduce<U, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int beg_idx = threadIdx.x * feature_size + blockIdx.x + col_offset;
  int end_idx = batch_size * feature_size + blockIdx.x + col_offset;
  int stride = BlockDim * feature_size;
  U d_scale_or_d_bias_partial = static_cast<U>(0);

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val =
        static_cast<U>(real_sqrt(static_cast<float>(var[row_idx]) + epsilon));
    if (HasDScale) {
      d_scale_or_d_bias_partial += static_cast<U>(d_y[i]) *
                                   (static_cast<U>(x[i]) - mean[row_idx]) /
                                   var_val;
    } else {  // d_bias != nullptr
      d_scale_or_d_bias_partial += static_cast<U>(d_y[i]);
    }

    if (HasDx) {
      if (scale != nullptr) {
        d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) *
                                scale[blockIdx.x + col_offset] / var_val);
      } else {
        d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) / var_val);
      }
    }
  }

  d_scale_or_d_bias_partial =
      BlockReduce(temp_storage).Reduce(d_scale_or_d_bias_partial, hipcub::Sum());

  if (threadIdx.x == 0) {
    if (HasDScale) {
      d_scale[blockIdx.x + col_offset] = d_scale_or_d_bias_partial;
    } else {
      d_bias[blockIdx.x + col_offset] = d_scale_or_d_bias_partial;
    }
  }
}

template <typename T, typename U, int BlockDim>
__global__ void LayerNormBackwardPostProcessToCalculateDX(const T *x, T *d_x,
                                                          const U *mean,
                                                          const U *var,
                                                          float epsilon,
                                                          int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ U d_x_reduce_tmp[2];

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  U block_mean = mean[blockIdx.x];
  U block_var = var[blockIdx.x];
  U d_x_mean_partial = static_cast<U>(0), d_x_var_partial = static_cast<U>(0);
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x_mean_partial += static_cast<U>(d_x[i]);
    d_x_var_partial +=
        static_cast<U>(d_x[i]) * (static_cast<U>(x[i]) - block_mean);
  }

  auto pair =
      BlockReduce(temp_storage)
          .Reduce(PairForLayerNorm<U>(d_x_mean_partial, d_x_var_partial),
                  PairForLayerNormAddFunctor<U>());

  if (threadIdx.x == 0) {
    d_x_reduce_tmp[0] = static_cast<float>(pair.first_) / feature_size;
    d_x_reduce_tmp[1] =
        static_cast<float>(pair.second_) /
        (feature_size * (static_cast<float>(block_var) + epsilon));
  }
  __syncthreads();

  d_x_mean_partial = d_x_reduce_tmp[0];
  d_x_var_partial = d_x_reduce_tmp[1];
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x[i] -= static_cast<T>(d_x_mean_partial);
    d_x[i] -=
        static_cast<T>((static_cast<U>(x[i]) - block_mean) * d_x_var_partial);
  }
}

// Here, we only calculate d_x
template <typename T, typename U, int BlockDim>
__global__ void LayerNormBackwardGradientOnlyDX(const T *x, const T *d_y,
                                                T *d_x, const U *mean,
                                                const U *var, const U *scale,
                                                float epsilon,
                                                int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ U d_x_reduce_tmp[2];

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  U block_mean = mean[blockIdx.x], block_var = var[blockIdx.x];
  U d_x_mean_partial = static_cast<U>(0), d_x_var_partial = static_cast<U>(0);
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    auto var_val =
        static_cast<U>(real_sqrt(static_cast<float>(block_var) + epsilon));
    if (scale != nullptr) {
      int col_idx = i % feature_size;
      d_x[i] =
          static_cast<T>(static_cast<U>(d_y[i]) * scale[col_idx] / var_val);
    } else {
      d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) / var_val);
    }
    d_x_mean_partial += static_cast<U>(d_x[i]);
    d_x_var_partial +=
        static_cast<U>(d_x[i]) * (static_cast<U>(x[i]) - block_mean);
  }

  auto pair =
      BlockReduce(temp_storage)
          .Reduce(PairForLayerNorm<U>(d_x_mean_partial, d_x_var_partial),
                  PairForLayerNormAddFunctor<U>());

  if (threadIdx.x == 0) {
    d_x_reduce_tmp[0] = static_cast<float>(pair.first_) / feature_size;
    d_x_reduce_tmp[1] =
        static_cast<float>(pair.second_) /
        (feature_size * (static_cast<float>(block_var) + epsilon));
  }
  __syncthreads();

  d_x_mean_partial = d_x_reduce_tmp[0];
  d_x_var_partial = d_x_reduce_tmp[1];
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x[i] -= static_cast<T>(d_x_mean_partial);
    d_x[i] -=
        static_cast<T>((static_cast<U>(x[i]) - block_mean) * d_x_var_partial);
  }
}

template <typename T, typename U>
__global__ void LayerNormBackwardWhenBatchSizeIsOne(
    const T *x, const T *d_y, T *d_x, U *d_scale, U *d_bias, const U *mean,
    const U *var, const U *scale, float epsilon, int feature_size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < feature_size) {
    auto var_val =
        static_cast<U>(real_sqrt(static_cast<float>(var[idx]) + epsilon));
    if (d_x != nullptr) {
      if (d_scale == nullptr) {
        d_x[idx] = static_cast<T>(static_cast<U>(d_y[idx]) / var_val);
      } else {
        d_x[idx] =
            static_cast<T>(static_cast<U>(d_y[idx]) * scale[idx] / var_val);
      }
    }

    if (d_scale != nullptr) {
      d_scale[idx] = static_cast<U>(d_y[idx]) *
                     (static_cast<U>(x[idx]) - mean[idx]) / var_val;
    }

    if (d_bias != nullptr) d_bias[idx] = static_cast<U>(d_y[idx]);
  }
}

template <typename T, typename U>
static void LayerNormBackward(const T *x, const T *d_y, const U *scale,
                              const U *mean, const U *var, T *d_x, U *d_scale,
                              U *d_bias, float epsilon, int batch_size,
                              int feature_size, hipStream_t stream) {
  const int kMaxBlockDim = 512;
  const int kMaxBlockNum = 128;
  int gradient_flag = ((d_x != nullptr ? 1 : 0) << 2) |
                      ((d_scale != nullptr ? 1 : 0) << 1) |
                      ((d_bias != nullptr ? 1 : 0));
  if (gradient_flag == 0) return;

  if (batch_size == 1) {
    LayerNormBackwardWhenBatchSizeIsOne<
        T, U><<<(feature_size + kMaxBlockDim - 1) / kMaxBlockDim, kMaxBlockDim,
                0, stream>>>(x, d_y, d_x, d_scale, d_bias, mean, var, scale,
                             epsilon, feature_size);

    if (d_x != nullptr) {
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardPostProcessToCalculateDX<
                             T, U, kBlockDim><<<1, kBlockDim, 0, stream>>>(
            x, d_x, mean, var, epsilon, feature_size));
      }
    }
    return;
  }

  auto block_dim = GetDesiredBlockDim(batch_size);
  switch (gradient_flag) {
    case 1:  // d_x == nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, false,
                false><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      break;
    case 2:  // d_x == nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, false,
                true><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      break;
    case 3:  // d_x == nullptr, d_scale != nulptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientAll<
                T, U, kBlockDim, false><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      break;
    case 4:  // d_x != nullptr, d_scale == nullptr, d_bias == nullptr
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientOnlyDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_y, d_x, mean, var, scale, epsilon, feature_size));
      }
      break;
    case 5:  // d_x != nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, true,
                false><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    case 6:  // d_x != nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, true,
                true><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    case 7:  // d_x != nullptr, d_scale != nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientAll<
                T, U, kBlockDim, true><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    default:
      break;
  }
}

template <typename T>
void LayerNormDirectCUDAFunctor<T>::operator()(hipStream_t stream,
                                               const T *input,
                                               std::vector<int> input_shape,
                                               const T *bias, const T *scale,
                                               T *output, T *mean, T *variance,
                                               int begin_norm_axis, float eps) {
  const auto x_dims = framework::make_ddim(input_shape);
  auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
  int batch_size = static_cast<int>(matrix_dim[0]);
  int feature_size = static_cast<int>(matrix_dim[1]);
  switch (GetDesiredBlockDim(feature_size)) {
    FIXED_BLOCK_DIM_CASE(
        LayerNormForward<T, T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
            input, scale, bias, output, mean, variance, eps, feature_size));
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Product from begin_norm_axis to end in layer_norm must be larger "
          "than 1"));
      break;
  }
}

template <typename T>
class LayerNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *x = ctx.Input<Tensor>("X");

    auto *y = ctx.Output<Tensor>("Y");
    auto *mean = ctx.Output<Tensor>("Mean");
    auto *var = ctx.Output<Tensor>("Variance");
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");

    const auto x_dims = x->dims();
    auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    auto *mean_data = mean->mutable_data<LayerNormParamType<T>>(ctx.GetPlace());
    auto *var_data = var->mutable_data<LayerNormParamType<T>>(ctx.GetPlace());
    auto *scale_data =
        (scale == nullptr ? nullptr : scale->data<LayerNormParamType<T>>());
    auto *bias_data =
        (bias == nullptr ? nullptr : bias->data<LayerNormParamType<T>>());

    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    switch (GetDesiredBlockDim(feature_size)) {
      FIXED_BLOCK_DIM_CASE(
          LayerNormForward<T, LayerNormParamType<T>,
                           kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
              x_data, scale_data, bias_data, y_data, mean_data, var_data,
              epsilon, feature_size));
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "Product from begin_norm_axis to end must be larger than 1"));
        break;
    }
  }
};

template <typename T>
class LayerNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    // d_x, d_scale, d_bias may be nullptr
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *x = ctx.Input<Tensor>("X");
    auto *mean = ctx.Input<Tensor>("Mean");
    auto *var = ctx.Input<Tensor>("Variance");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    auto *x_data = x->data<T>();
    auto *d_y_data = d_y->data<T>();
    auto *mean_data = mean->data<U>();
    auto *var_data = var->data<U>();

    auto *scale_data = (scale == nullptr ? nullptr : scale->data<U>());
    auto *d_scale_data =
        (d_scale == nullptr ? nullptr
                            : d_scale->mutable_data<U>(ctx.GetPlace()));
    auto *d_bias_data =
        (d_bias == nullptr ? nullptr : d_bias->mutable_data<U>(ctx.GetPlace()));
    auto *d_x_data =
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace()));

    const auto &x_dims = x->dims();
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    LayerNormBackward<T, U>(x_data, d_y_data, scale_data, mean_data, var_data,
                            d_x_data, d_scale_data, d_bias_data, epsilon,
                            batch_size, feature_size, stream);
  }
};

template class LayerNormDirectCUDAFunctor<float>;

#undef FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE
#undef FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE
#undef FIXED_BLOCK_DIM_CASE_BASE
#undef FIXED_BLOCK_DIM_CASE
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>);
