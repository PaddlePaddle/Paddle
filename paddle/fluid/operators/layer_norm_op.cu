#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/fluid/operators/layer_norm_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
void LayerNormDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                               const T *input,
                                               std::vector<int> input_shape,
                                               const T *bias, const T *scale,
                                               T *output, T *mean, T *variance,
                                               int begin_norm_axis, float eps) {
  const auto x_dims = phi::make_ddim(input_shape);
  auto matrix_dim = phi::flatten_to_2d(x_dims, begin_norm_axis);
  int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
  int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);
  switch (GetDesiredBlockDim(feature_size)) {
    FIXED_BLOCK_DIM_CASE(
        LayerNormForward<T, T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
            input, scale, bias, output, mean, variance, eps, feature_size));
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Product from begin_norm_axis to end in layer_norm must be larger "
          "than 1"));
      break;
  }
}

template <typename T>
class LayerNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *x = ctx.Input<Tensor>("X");

    auto *y = ctx.Output<Tensor>("Y");
    auto *mean = ctx.Output<Tensor>("Mean");
    auto *var = ctx.Output<Tensor>("Variance");
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");

    const auto x_dims = x->dims();
    auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    auto *mean_data = mean->mutable_data<U>(ctx.GetPlace());
    auto *var_data = var->mutable_data<U>(ctx.GetPlace());

    auto *void_scale_data = (scale == nullptr ? nullptr : scale->data());
    auto *void_bias_data = (bias == nullptr ? nullptr : bias->data());

    framework::proto::VarType::Type x_dtype =
        framework::TransToProtoVarType(x->dtype());
    framework::proto::VarType::Type scale_bias_dtype;
    if (void_scale_data != nullptr) {
      scale_bias_dtype = framework::TransToProtoVarType(scale->dtype());
      if (void_bias_data != nullptr) {
        PADDLE_ENFORCE_EQ(scale_bias_dtype,
                          framework::TransToProtoVarType(bias->dtype()),
                          platform::errors::InvalidArgument(
                              "Thie Scale and Bias of layer_norm op "
                              "should have the same data type."));
      }
    } else {
      scale_bias_dtype = (void_bias_data != nullptr
                              ? framework::TransToProtoVarType(bias->dtype())
                              : x_dtype);
    }

    bool is_scale_bias_same_dtype_with_x = x_dtype == scale_bias_dtype;
    if (!is_scale_bias_same_dtype_with_x) {
      PADDLE_ENFORCE_EQ(scale_bias_dtype,
                        framework::DataTypeTrait<U>::DataType(),
                        platform::errors::InvalidArgument(
                            "Unsupported data type of Scale and Bias: %s",
                            framework::DataTypeToString(scale_bias_dtype)));
    }

    auto matrix_dim = phi::flatten_to_2d(x_dims, begin_norm_axis);
    int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
    int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

#define PADDLE_LAUNCH_LAYERNORM_FWD(ScaleBiasT, IsScaleBiasSameDTypeWithX) \
  do {                                                                     \
    switch (GetDesiredBlockDim(feature_size)) {                            \
      FIXED_BLOCK_DIM_CASE(                                                \
          LayerNormForward<T, U, kBlockDim, IsScaleBiasSameDTypeWithX><<<  \
              batch_size, kBlockDim, 0, stream>>>(                         \
              x_data, static_cast<const ScaleBiasT *>(void_scale_data),    \
              static_cast<const ScaleBiasT *>(void_bias_data), y_data,     \
              mean_data, var_data, epsilon, feature_size));                \
      default:                                                             \
        PADDLE_THROW(platform::errors::InvalidArgument(                    \
            "Product from begin_norm_axis to end must be larger than 1")); \
        break;                                                             \
    }                                                                      \
  } while (0)

#ifdef PADDLE_WITH_CUDA
    bool can_call_1024_kernel = false;
    if (feature_size == 1024 && scale != nullptr && bias != nullptr) {
      can_call_1024_kernel = true;
    }
    if (can_call_1024_kernel) {
      const int WARPS_M = 4;
      const int WARPS_N = 1;
      const int THREADS_PER_WARP = 32;
      const int BYTES_PER_LDG = 16;
      const int VecSize = BYTES_PER_LDG / sizeof(T);

      const int THREADS_PER_CTA = WARPS_N * THREADS_PER_WARP * WARPS_M;
      const int ROWS_PER_CTA = WARPS_M;

      const int grid = static_cast<int>(
          std::ceil(batch_size / static_cast<float>(ROWS_PER_CTA)));
      if (is_scale_bias_same_dtype_with_x) {
        ln_fwd_1024_kernel<T, U, T, VecSize, WARPS_M, WARPS_N,
                           BYTES_PER_LDG><<<grid, THREADS_PER_CTA, 0, stream>>>(
            batch_size, feature_size, epsilon, x_data,
            static_cast<const T *>(void_scale_data),
            static_cast<const T *>(void_bias_data), mean_data, var_data,
            y_data);
      } else {
        ln_fwd_1024_kernel<T, U, U, VecSize, WARPS_M, WARPS_N,
                           BYTES_PER_LDG><<<grid, THREADS_PER_CTA, 0, stream>>>(
            batch_size, feature_size, epsilon, x_data,
            static_cast<const U *>(void_scale_data),
            static_cast<const U *>(void_bias_data), mean_data, var_data,
            y_data);
      }
    } else {
#endif
      if (is_scale_bias_same_dtype_with_x) {
        PADDLE_LAUNCH_LAYERNORM_FWD(T, true);
      } else {
        PADDLE_LAUNCH_LAYERNORM_FWD(U, false);
      }
#ifdef PADDLE_WITH_CUDA
    }
#endif

#undef PADDLE_LAUNCH_LAYERNORM_FWD
  }
};

template <typename T>
class LayerNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    // d_x, d_scale, d_bias may be nullptr
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *x = ctx.Input<Tensor>("X");
    auto *mean = ctx.Input<Tensor>("Mean");
    auto *var = ctx.Input<Tensor>("Variance");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    const auto &x_dims = x->dims();
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    auto matrix_dim = phi::flatten_to_2d(x_dims, begin_norm_axis);
    int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
    int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);

    auto *x_data = x->data<T>();
    auto *d_y_data = d_y->data<T>();

    auto *mean_data = mean->data<U>();
    auto *var_data = var->data<U>();

    auto *d_x_data =
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace()));

    framework::proto::VarType::Type x_dtype =
        framework::TransToProtoVarType(x->dtype());
    framework::proto::VarType::Type scale_bias_dtype;
    if (scale != nullptr) {
      scale_bias_dtype = framework::TransToProtoVarType(scale->dtype());
    } else {
      // FIXME(zengjinle): do not find a better way to get the right
      // data type of the d_scale and d_bias if scale == nullptr.
      auto *bias = ctx.Input<Tensor>("Bias");
      if (bias != nullptr) {
        scale_bias_dtype = framework::TransToProtoVarType(bias->dtype());
      } else {
        scale_bias_dtype = x_dtype;
      }
    }

#define PADDLE_LAUNCH_LAYERNORM_BWD(ScaleBiasT, IsScaleBiasSameDTypeWithX) \
  do {                                                                     \
    auto *scale_data =                                                     \
        (scale == nullptr ? nullptr : scale->data<ScaleBiasT>());          \
    auto *d_scale_data =                                                   \
        (d_scale == nullptr ? nullptr : d_scale->mutable_data<ScaleBiasT>( \
                                            ctx.GetPlace()));              \
    auto *d_bias_data =                                                    \
        (d_bias == nullptr ? nullptr : d_bias->mutable_data<ScaleBiasT>(   \
                                           ctx.GetPlace()));               \
    auto *d_x_data =                                                       \
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace())); \
    LayerNormBackward<T, U, IsScaleBiasSameDTypeWithX>(                    \
        x_data, d_y_data, scale_data, mean_data, var_data, d_x_data,       \
        d_scale_data, d_bias_data, epsilon, batch_size, feature_size,      \
        ctx.cuda_device_context());                                        \
  } while (0)

    if (scale_bias_dtype == x_dtype) {
      PADDLE_LAUNCH_LAYERNORM_BWD(T, true);
    } else {
      PADDLE_LAUNCH_LAYERNORM_BWD(U, false);
    }

#undef PADDLE_LAUNCH_LAYERNORM_BWD
  }
};

template class LayerNormDirectCUDAFunctor<float>;
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>);
#elif CUDNN_VERSION_MIN(8, 1, 0)
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::bfloat16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::bfloat16>);
#else
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>);
#endif
