#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hipcub/hipcub.hpp>
#include "paddle/fluid/operators/layer_norm_op.h"

namespace paddle {
namespace operators {

inline static int GetDesiredBlockDim(int block_dim) {
  const int kMaxBlockDim = 512;
  return block_dim >= kMaxBlockDim
             ? kMaxBlockDim
             : (1 << (static_cast<int>(std::log2f(block_dim))));
}

#define FIXED_BLOCK_DIM_CASE_BASE(log2_block_dim, ...)  \
  case (1 << (log2_block_dim)): {                       \
    constexpr auto kBlockDim = (1 << (log2_block_dim)); \
    __VA_ARGS__;                                        \
  } break

#define FIXED_BLOCK_DIM_CASE(...)              \
  FIXED_BLOCK_DIM_CASE_BASE(9, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(8, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(7, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(6, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(5, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(4, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(3, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(2, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(1, ##__VA_ARGS__)

template <typename T, int BlockDim>
__global__ void LayerNormForward(const T *x, const T *scale, const T *bias,
                                 T *y, T *mean, T *var, float epsilon,
                                 int feature_size) {
  using BlockReduce = hipcub::BlockReduce<T, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  // Step 1: Reduce to calculate mean
  T mean_val = static_cast<T>(0);
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    mean_val += x[beg_idx];
  }
  mean_val = BlockReduce(temp_storage).Reduce(mean_val, hipcub::Sum());
  if (threadIdx.x == 0) mean[blockIdx.x] = mean_val / feature_size;
  __syncthreads();
  mean_val = mean[blockIdx.x];

  // Step 2: Reduce to calculate var
  T var_val = static_cast<T>(0);
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T tmp = x[i] - mean_val;
    var_val += (tmp * tmp);
  }
  var_val = BlockReduce(temp_storage).Reduce(var_val, hipcub::Sum());
  if (threadIdx.x == 0) {
    var[blockIdx.x] = var_val / feature_size;
  }
  __syncthreads();
  var_val = static_cast<T>(sqrt(var[blockIdx.x] + epsilon));

  // Step 3: Calculate y
  if (scale != nullptr) {
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = scale[j] * (x[i] - mean_val) / var_val + bias[j];
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = scale[j] * (x[i] - mean_val) / var_val;
      }
    }
  } else {  // scale == nullptr
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = (x[i] - mean_val) / var_val + bias[j];
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = (x[i] - mean_val) / var_val;
      }
    }
  }
}

template <typename T>
struct Pair {
  __device__ __forceinline__ Pair() {}
  __device__ __forceinline__ Pair(const T &first, const T &second)
      : first_(first), second_(second) {}

  T first_;
  T second_;
};

template <typename T>
struct PairAddFunctor {
  __device__ __forceinline__ Pair<T> operator()(const Pair<T> &p1,
                                                const Pair<T> &p2) {
    return Pair<T>(p1.first_ + p2.first_, p1.second_ + p2.second_);
  }
};

// Make sure that d_scale != nullptr && d_bias != nullptr
// Since d_scale != nullptr, scale would not be nullptr
template <typename T, int BlockDim, bool HasDx>
__global__ void LayerNormBackwardGradientAll(const T *x, const T *d_y,
                                             T *d_scale, T *d_bias, T *d_x,
                                             const T *mean, const T *var,
                                             const T *scale, float epsilon,
                                             int batch_size, int feature_size) {
  using BlockReduce = hipcub::BlockReduce<Pair<T>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = threadIdx.x * feature_size + blockIdx.x;
  int end_idx = batch_size * feature_size + blockIdx.x;
  int stride = BlockDim * feature_size;
  T d_scale_partial = 0, d_bias_partial = 0;

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val = static_cast<T>(sqrt(var[row_idx] + epsilon));
    d_scale_partial += d_y[i] * (x[i] - mean[row_idx]) / var_val;
    d_bias_partial += d_y[i];
    if (HasDx) d_x[i] = d_y[i] * scale[blockIdx.x] / var_val;
  }

  auto pair = BlockReduce(temp_storage)
                  .Reduce(Pair<T>(d_scale_partial, d_bias_partial),
                          PairAddFunctor<T>());

  if (threadIdx.x == 0) {
    d_scale[blockIdx.x] = pair.first_;
    d_bias[blockIdx.x] = pair.second_;
  }
}

// Make sure that there is only one true expression: d_scale != nullptr
// or d_bias != nullptr
// Notice: scale may be nullptr
template <typename T, int BlockDim, bool HasDx, bool HasDScale>
__global__ void LayerNormBackwardGradientScaleOrBias(
    const T *x, const T *d_y, T *d_scale, T *d_bias, T *d_x, const T *mean,
    const T *var, const T *scale, float epsilon, int batch_size,
    int feature_size) {
  using BlockReduce = hipcub::BlockReduce<T, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int beg_idx = threadIdx.x * feature_size + blockIdx.x;
  int end_idx = batch_size * feature_size + blockIdx.x;
  int stride = BlockDim * feature_size;
  T d_scale_or_d_bias_partial = 0;

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val = static_cast<T>(sqrt(var[row_idx] + epsilon));
    if (HasDScale) {
      d_scale_or_d_bias_partial += d_y[i] * (x[i] - mean[row_idx]) / var_val;
    } else {  // d_bias != nullptr
      d_scale_or_d_bias_partial += d_y[i];
    }

    if (HasDx) {
      if (scale != nullptr)
        d_x[i] = d_y[i] * scale[blockIdx.x] / var_val;
      else
        d_x[i] = d_y[i] / var_val;
    }
  }

  d_scale_or_d_bias_partial =
      BlockReduce(temp_storage).Reduce(d_scale_or_d_bias_partial, hipcub::Sum());

  if (threadIdx.x == 0) {
    if (HasDScale) {
      d_scale[blockIdx.x] = d_scale_or_d_bias_partial;
    } else {
      d_bias[blockIdx.x] = d_scale_or_d_bias_partial;
    }
  }
}

// Here, we only calculate d_x
template <typename T>
__global__ void LayerNormBackwardGradientOnlyX(const T *d_y, T *d_x,
                                               const T *var, const T *scale,
                                               float epsilon, int batch_size,
                                               int feature_size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < batch_size * feature_size) {
    int row_idx = idx / feature_size;
    auto var_val = static_cast<T>(sqrt(var[row_idx] + epsilon));
    if (scale != nullptr) {
      int col_idx = idx % feature_size;
      d_x[idx] = d_y[idx] * scale[col_idx] / var_val;
    } else {
      d_x[idx] = d_y[idx] / var_val;
    }
  }
}

template <typename T>
__global__ void LayerNormBackwardWhenBatchSizeIsOne(
    const T *x, const T *d_y, T *d_x, T *d_scale, T *d_bias, const T *mean,
    const T *var, const T *scale, float epsilon, int feature_size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < feature_size) {
    auto var_val = static_cast<T>(sqrt(var[idx] + epsilon));
    if (d_x != nullptr) {
      if (d_scale == nullptr)
        d_x[idx] = d_y[idx] / var_val;
      else
        d_x[idx] = d_y[idx] * scale[idx] / var_val;
    }
    if (d_scale != nullptr)
      d_scale[idx] = d_y[idx] * (x[idx] - mean[idx]) / var_val;
    if (d_bias != nullptr) d_bias[idx] = d_y[idx];
  }
}

template <typename T>
static void LayerNormBackward(const T *x, const T *d_y, const T *scale,
                              const T *mean, const T *var, T *d_x, T *d_scale,
                              T *d_bias, float epsilon, int batch_size,
                              int feature_size, hipStream_t stream) {
  const int kMaxBlockDim = 512;
  int gradient_flag = (static_cast<int>(d_x != nullptr) << 2) |
                      (static_cast<int>(d_scale != nullptr) << 1) |
                      (static_cast<int>(d_bias != nullptr));
  if (gradient_flag == 0) return;

  if (batch_size == 1) {
    LayerNormBackwardWhenBatchSizeIsOne<
        T><<<(feature_size + kMaxBlockDim - 1) / kMaxBlockDim, kMaxBlockDim, 0,
             stream>>>(x, d_y, d_x, d_scale, d_bias, mean, var, scale, epsilon,
                       feature_size);
    return;
  }

  auto block_dim = GetDesiredBlockDim(batch_size);
  switch (gradient_flag) {
    case 1:  // d_x == nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, false,
                             false><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      break;
    case 2:  // d_x == nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, false,
                             true><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      break;
    case 3:  // d_x == nullptr, d_scale != nulptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientAll<
                T, kBlockDim, false><<<feature_size, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size));
      }
      break;
    case 4:  // d_x != nullptr, d_scale == nullptr, d_bias == nullptr
      LayerNormBackwardGradientOnlyX<
          T><<<(batch_size * feature_size + kMaxBlockDim - 1) / kMaxBlockDim,
               kMaxBlockDim, 0, stream>>>(d_y, d_x, var, scale, epsilon,
                                          batch_size, feature_size);
      break;
    case 5:  // d_x != nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, true,
                             false><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      break;
    case 6:  // d_x != nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardGradientScaleOrBias<
                             T, kBlockDim, true,
                             true><<<feature_size, kBlockDim, 0, stream>>>(
            x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon, batch_size,
            feature_size));
      }
      break;
    case 7:  // d_x != nullptr, d_scale != nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientAll<
                T, kBlockDim, true><<<feature_size, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size));
      }
      break;
    default:
      break;
  }
}

template <typename T>
class LayerNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *x = ctx.Input<Tensor>("X");

    auto *y = ctx.Output<Tensor>("Y");
    auto *mean = ctx.Output<Tensor>("Mean");
    auto *var = ctx.Output<Tensor>("Variance");
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");

    const auto x_dims = x->dims();
    auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    auto *mean_data = mean->mutable_data<T>(ctx.GetPlace());
    auto *var_data = var->mutable_data<T>(ctx.GetPlace());
    auto *scale_data = (scale == nullptr ? nullptr : scale->data<T>());
    auto *bias_data = (bias == nullptr ? nullptr : bias->data<T>());

    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    switch (GetDesiredBlockDim(feature_size)) {
      FIXED_BLOCK_DIM_CASE(
          LayerNormForward<T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
              x_data, scale_data, bias_data, y_data, mean_data, var_data,
              epsilon, feature_size));
      default:
        PADDLE_THROW(
            "Product from begin_norm_axis to end must be larger than 1");
        break;
    }
  }
};

template <typename T>
class LayerNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const float epsilon = ctx.Attr<float>("epsilon");
    // d_x, d_scale, d_bias may be nullptr
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *x = ctx.Input<Tensor>("X");
    auto *mean = ctx.Input<Tensor>("Mean");
    auto *var = ctx.Input<Tensor>("Variance");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    auto *x_data = x->data<T>();
    auto *d_y_data = d_y->data<T>();
    auto *mean_data = mean->data<T>();
    auto *var_data = var->data<T>();
    auto *scale_data = (scale == nullptr ? nullptr : scale->data<T>());
    auto *d_scale_data =
        (d_scale == nullptr ? nullptr
                            : d_scale->mutable_data<T>(ctx.GetPlace()));
    auto *d_bias_data =
        (d_bias == nullptr ? nullptr : d_bias->mutable_data<T>(ctx.GetPlace()));
    auto *d_x_data =
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace()));

    const auto &x_dims = x->dims();
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    LayerNormBackward<T>(x_data, d_y_data, scale_data, mean_data, var_data,
                         d_x_data, d_scale_data, d_bias_data, epsilon,
                         batch_size, feature_size, stream);
  }
};

#undef FIXED_BLOCK_DIM_CASE_BASE
#undef FIXED_BLOCK_DIM_CASE
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>);
