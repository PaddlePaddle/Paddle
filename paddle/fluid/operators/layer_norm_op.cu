/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/fluid/operators/layer_norm_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
void LayerNormDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                               const T *input,
                                               std::vector<int> input_shape,
                                               const T *bias, const T *scale,
                                               T *output, T *mean, T *variance,
                                               int begin_norm_axis, float eps) {
  const auto x_dims = framework::make_ddim(input_shape);
  auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
  int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
  int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);
  switch (GetDesiredBlockDim(feature_size)) {
    FIXED_BLOCK_DIM_CASE(
        LayerNormForward<T, T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
            input, scale, bias, output, mean, variance, eps, feature_size));
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Product from begin_norm_axis to end in layer_norm must be larger "
          "than 1"));
      break;
  }
}

template <typename T>
class LayerNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *x = ctx.Input<Tensor>("X");

    auto *y = ctx.Output<Tensor>("Y");
    auto *mean = ctx.Output<Tensor>("Mean");
    auto *var = ctx.Output<Tensor>("Variance");
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    const auto x_dims = x->dims();
    auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    auto *mean_data = mean->mutable_data<U>(ctx.GetPlace());
    auto *var_data = var->mutable_data<U>(ctx.GetPlace());
    auto *scale_data = (scale == nullptr ? nullptr : scale->data<T>());
    auto *bias_data = (bias == nullptr ? nullptr : bias->data<T>());

    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
    int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    switch (GetDesiredBlockDim(feature_size)) {
      FIXED_BLOCK_DIM_CASE(
          LayerNormForward<T, U,
                           kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
              x_data, scale_data, bias_data, y_data, mean_data, var_data,
              epsilon, feature_size));
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "Product from begin_norm_axis to end must be larger than 1"));
        break;
    }
  }
};

template <typename T>
class LayerNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    // d_x, d_scale, d_bias may be nullptr
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *x = ctx.Input<Tensor>("X");
    auto *mean = ctx.Input<Tensor>("Mean");
    auto *var = ctx.Input<Tensor>("Variance");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    auto *x_data = x->data<T>();
    auto *d_y_data = d_y->data<T>();
    auto *mean_data = mean->data<U>();
    auto *var_data = var->data<U>();

    auto *scale_data = (scale == nullptr ? nullptr : scale->data<T>());
    auto *d_scale_data =
        (d_scale == nullptr ? nullptr
                            : d_scale->mutable_data<T>(ctx.GetPlace()));
    auto *d_bias_data =
        (d_bias == nullptr ? nullptr : d_bias->mutable_data<T>(ctx.GetPlace()));
    auto *d_x_data =
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace()));

    const auto &x_dims = x->dims();
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
    int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);

    LayerNormBackward<T, U>(x_data, d_y_data, scale_data, mean_data, var_data,
                            d_x_data, d_scale_data, d_bias_data, epsilon,
                            batch_size, feature_size,
                            ctx.cuda_device_context());
  }
};

template class LayerNormDirectCUDAFunctor<float>;
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>);
#else
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>);
#endif
