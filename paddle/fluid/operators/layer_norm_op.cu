#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hipcub/hipcub.hpp>
#include <memory>
#include <vector>

#include "paddle/fluid/framework/ddim.h"
#include "paddle/fluid/operators/layer_norm_op.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using LayerNormParamType = typename CudnnDataType<T>::BatchNormParamType;

inline static int GetDesiredBlockDim(int block_dim) {
  const int kMaxBlockDim = 512;
  return block_dim >= kMaxBlockDim
             ? kMaxBlockDim
             : (1 << (static_cast<int>(std::log2f(block_dim))));
}

#define FIXED_BLOCK_DIM_CASE_BASE(log2_block_dim, ...)  \
  case (1 << (log2_block_dim)): {                       \
    constexpr auto kBlockDim = (1 << (log2_block_dim)); \
    __VA_ARGS__;                                        \
  } break

#define FIXED_BLOCK_DIM_CASE(...)              \
  FIXED_BLOCK_DIM_CASE_BASE(9, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(8, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(7, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(6, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(5, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(4, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(3, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(2, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(1, ##__VA_ARGS__)

#define FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(                             \
    log2_block_dim, feature_size, kMaxBlockNum, ...)                           \
  case (1 << (log2_block_dim)): {                                              \
    for (int i = 0; i < std::ceil(feature_size / (1.0 * kMaxBlockNum)); i++) { \
      int col_offset = i * kMaxBlockNum;                                       \
      int block_num = std::min(feature_size - col_offset, kMaxBlockNum);       \
      constexpr auto kBlockDim = (1 << (log2_block_dim));                      \
      __VA_ARGS__;                                                             \
    }                                                                          \
  } break

#define FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(feature_size, kMaxBlockNum, ...) \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(9, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(8, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(7, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(6, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(5, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(4, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(3, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(2, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__);                   \
  FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE(1, feature_size, kMaxBlockNum,    \
                                            ##__VA_ARGS__)

static __device__ __forceinline__ float real_sqrt(float x) { return sqrtf(x); }
static __device__ __forceinline__ double real_sqrt(double x) { return sqrt(x); }

template <typename T>
struct PairForLayerNorm {
  __device__ __forceinline__ PairForLayerNorm() {}
  __device__ __forceinline__ PairForLayerNorm(const T &first, const T &second)
      : first_(first), second_(second) {}

  T first_;
  T second_;
};

template <typename T>
struct PairForLayerNormAddFunctor {
  __device__ __forceinline__ PairForLayerNorm<T> operator()(
      const PairForLayerNorm<T> &p1, const PairForLayerNorm<T> &p2) {
    return PairForLayerNorm<T>(p1.first_ + p2.first_, p1.second_ + p2.second_);
  }
};

template <typename T>
__inline__ __device__ T rsqrt_(const T val) {
  return static_cast<T>(1) / sqrt(val);
}

template <>
__inline__ __device__ float rsqrt_(const float val) {
  return rsqrtf(val);
}

template <>
__inline__ __device__ double rsqrt_(const double val) {
  return rsqrt(val);
}

#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
template <>
__inline__ __device__ half rsqrt_(const half val) {
  return hrsqrt(val);
}
#endif

template <typename T, typename U, int BlockDim>
__global__ void LayerNormForward(const T *x, const U *scale, const U *bias,
                                 T *y, U *mean, U *var, float epsilon,
                                 int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ U mean_share;
  __shared__ U var_share;

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  // Step 1: Reduce to calculate mean and var
  U mean_val = 0;
  U var_val = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    U tmp = static_cast<U>(x[i]);
    mean_val += tmp;
    var_val += (tmp * tmp);
  }
  auto pair = BlockReduce(temp_storage)
                  .Reduce(PairForLayerNorm<U>(mean_val, var_val),
                          PairForLayerNormAddFunctor<U>());
  if (threadIdx.x == 0) {
    auto tmp = pair.first_ / feature_size;
    mean[blockIdx.x] = mean_share = static_cast<U>(tmp);
    var[blockIdx.x] = var_share =
        static_cast<U>(pair.second_ / feature_size - tmp * tmp);
  }
  __syncthreads();

  mean_val = mean_share;
  U invvar = rsqrt_<U>(var_share + static_cast<U>(epsilon));

  // Step 2: Calculate y
  if (scale != nullptr) {
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>(
            scale[j] * (static_cast<U>(x[i]) - mean_val) * invvar + bias[j]);
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>(scale[j] * (static_cast<U>(x[i]) - mean_val) *
                              invvar);
      }
    }
  } else {  // scale == nullptr
    if (bias != nullptr) {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>((static_cast<U>(x[i]) - mean_val) * invvar +
                              bias[j]);
      }
    } else {
      for (int i = beg_idx, j = threadIdx.x; i < end_idx;
           i += BlockDim, j += BlockDim) {
        y[i] = static_cast<T>((static_cast<U>(x[i]) - mean_val) * invvar);
      }
    }
  }
}

template <typename T, typename U, int VPT>
__inline__ __device__ void cuLoadAddStridedInputs(
    const int i1_block, const int thr_load_row_off, const int thr_load_col_off,
    const int i2_off, const int row_stride, U *warp_buf1, U *warp_buf2,
    const T *input, const T *dout, const int i1_end, const int n2,
    const U *__restrict__ mean, const U *__restrict__ var,
    const float epsilon) {
  const int i1 = i1_block + thr_load_row_off;
  if (i1 >= i1_end) return;
  U curr_mean = mean[i1];
  U curr_invvar = rsqrt_<U>(var[i1] + epsilon);
  for (int k = 0; k < VPT; ++k) {
    const int i2 = i2_off + k;
    const int load_idx = i1 * n2 + i2;
    const int write_idx = thr_load_row_off * row_stride + thr_load_col_off + k;
    if (i2 < n2) {
      U curr_input = static_cast<U>(input[load_idx]);
      U curr_dout = static_cast<U>(dout[load_idx]);
      warp_buf1[write_idx] += curr_dout;
      warp_buf2[write_idx] +=
          curr_dout * (curr_input - curr_mean) * curr_invvar;
    }
  }
}

template <typename T, typename U, int BDIMX, int BDIMY, int VPTX>
__global__ void LayerNormBackwardPartGradGammaBeta(
    const T *__restrict__ dout, const T *__restrict__ input, const int n1,
    const int n2, const U *__restrict__ mean, const U *__restrict__ var,
    float epsilon, U *part_grad_gamma, U *part_grad_beta) {
  // VPTX -> value per thread.x, BDIMX -> blockDim.x, BDIMY -> blockDim.y, BDIMX
  // -> blockDim.x
  // template for compile time optimizations

  constexpr int row_stride = BDIMX + 1;
  const int thr_load_col_off = (threadIdx.x * VPTX) & (BDIMX - 1);
  const int thr_load_row_off =
      (threadIdx.x * VPTX) / BDIMX + threadIdx.y * BDIMY;
  const int i2_off = blockIdx.x * BDIMX + thr_load_col_off;

  constexpr int shared_cap = (BDIMX * BDIMY > 2 * VPTX * BDIMY * row_stride)
                                 ? BDIMX * BDIMY
                                 : 2 * VPTX * BDIMY * row_stride;
  __shared__ U buf[shared_cap];

  U *warp_buf1 = reinterpret_cast<U *>(buf);
  U *warp_buf2 = warp_buf1 + VPTX * BDIMY * row_stride;

  for (int idx = threadIdx.y * blockDim.x + threadIdx.x;
       idx < 2 * VPTX * BDIMY * row_stride; idx += BDIMX * BDIMY) {
    buf[idx] = U(0);
  }
  __syncthreads();

  for (int i1_block = blockIdx.y * BDIMY * VPTX; i1_block < n1;
       i1_block += VPTX * BDIMY * gridDim.y) {
    cuLoadAddStridedInputs<T, U, VPTX>(
        i1_block, thr_load_row_off, thr_load_col_off, i2_off, row_stride,
        warp_buf1, warp_buf2, input, dout, n1, n2, mean, var, epsilon);
  }
  __syncthreads();

  // inter-warp reductions
  // sum within each warp
  U acc1 = U(0);
  U acc2 = U(0);
  for (int k = 0; k < VPTX; ++k) {
    int row1 = threadIdx.y + k * VPTX;
    int idx1 = row1 * row_stride + threadIdx.x;
    acc1 += warp_buf1[idx1];
    acc2 += warp_buf2[idx1];
  }
  warp_buf1[threadIdx.y * row_stride + threadIdx.x] = acc1;
  warp_buf2[threadIdx.y * row_stride + threadIdx.x] = acc2;
  __syncthreads();
  // sum all warps
  for (int offset = VPTX >> 1; offset > 1; offset >>= 1) {
    if (threadIdx.y < offset) {
      int row1 = threadIdx.y;
      int row2 = threadIdx.y + offset;
      int idx1 = row1 * row_stride + threadIdx.x;
      int idx2 = row2 * row_stride + threadIdx.x;
      warp_buf1[idx1] += warp_buf1[idx2];
      warp_buf2[idx1] += warp_buf2[idx2];
    }
    __syncthreads();
  }
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.y == 0 && i2 < n2) {
    int row1 = threadIdx.y;
    int row2 = threadIdx.y + 1;
    int idx1 = row1 * row_stride + threadIdx.x;
    int idx2 = row2 * row_stride + threadIdx.x;
    part_grad_beta[blockIdx.y * n2 + i2] = warp_buf1[idx1] + warp_buf1[idx2];
    part_grad_gamma[blockIdx.y * n2 + i2] = warp_buf2[idx1] + warp_buf2[idx2];
  }
}

template <typename T, typename U, int BDIMX, int BDIMY>
__global__ void LayerNormBackwardSumGradGammaBeta(
    const U *part_grad_gamma, const U *part_grad_beta, const int part_size,
    // const int n1, const int n2, T* grad_gamma, T* grad_beta) {
    const int n1, const int n2, U *grad_gamma, U *grad_beta) {
  // sum partial gradients for gamma and beta
  __shared__ U buf[BDIMX * BDIMY];
  int i2 = blockIdx.x * BDIMX + threadIdx.x;
  if (i2 < n2) {
    // each warp does sequential reductions until reduced part_size is num_warps
    int num_warp_reductions = part_size / BDIMY;
    U sum_gamma = U(0);
    U sum_beta = U(0);
    const U *part_grad_gamma_ptr =
        part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
    const U *part_grad_beta_ptr =
        part_grad_beta + threadIdx.y * num_warp_reductions * n2 + i2;
    for (int warp_offset = 0; warp_offset < num_warp_reductions;
         ++warp_offset) {
      sum_gamma += part_grad_gamma_ptr[warp_offset * n2];
      sum_beta += part_grad_beta_ptr[warp_offset * n2];
    }
    // inter-warp reductions
    constexpr int nbsize3 = BDIMX * BDIMY / 2;
    for (int offset = BDIMY / 2; offset >= 1; offset /= 2) {
      // top half write to shared memory
      if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
        const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
        buf[write_idx] = sum_gamma;
        buf[write_idx + nbsize3] = sum_beta;
      }
      __syncthreads();
      // bottom half sums
      if (threadIdx.y < offset) {
        const int read_idx = threadIdx.y * BDIMX + threadIdx.x;
        sum_gamma += buf[read_idx];
        sum_beta += buf[read_idx + nbsize3];
      }
      __syncthreads();
    }
    // write out fully summed gradients
    if (threadIdx.y == 0) {
      grad_gamma[i2] = sum_gamma;
      grad_beta[i2] = sum_beta;
    }
  }
}

template <typename T, typename U, int BDIMX, int BDIMY>
__global__ void LayerNormBackwardComputeGradInput(
    const T *__restrict__ dout, const T *__restrict__ input, const int n1,
    const int n2,
    // const U* __restrict__ mean, const U* __restrict__ var, const float
    // epsilon, const T* gamma,
    const U *__restrict__ mean, const U *__restrict__ var, const float epsilon,
    const U *gamma, T *grad_input) {
  for (auto i1 = blockIdx.y; i1 < n1; i1 += gridDim.y) {
    U sum_loss1 = U(0);
    U sum_loss2 = U(0);
    const U c_mean = mean[i1];
    const U c_invvar = rsqrt_<U>(var[i1] + epsilon);
    const T *k_input = input + i1 * n2;
    const T *k_dout = dout + i1 * n2;
    constexpr int numx = BDIMX * BDIMY;
    const int thrx = threadIdx.x + threadIdx.y * BDIMX;
    if (gamma != NULL) {
      int l = 4 * thrx;
      for (; l + 3 < n2; l += 4 * numx) {
        for (int k = 0; k < 4; ++k) {
          const U c_h = static_cast<U>(k_input[l + k]);
          const U c_loss = static_cast<U>(k_dout[l + k]);
          sum_loss1 += c_loss * gamma[l + k];
          sum_loss2 += c_loss * gamma[l + k] * (c_h - c_mean) * c_invvar;
        }
      }
      for (; l < n2; ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        sum_loss1 += c_loss * gamma[l];
        sum_loss2 += c_loss * gamma[l] * (c_h - c_mean) * c_invvar;
      }
    } else {
      int l = 4 * thrx;
      for (; l + 3 < n2; l += 4 * numx) {
        for (int k = 0; k < 4; ++k) {
          const U c_h = static_cast<U>(k_input[l + k]);
          const U c_loss = static_cast<U>(k_dout[l + k]);
          sum_loss1 += c_loss;
          sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
        }
      }
      for (; l < n2; ++l) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        sum_loss1 += c_loss;
        sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
      }
    }
    // intra-warp reductions
    for (int mask = BDIMX / 2; mask > 0; mask /= 2) {
      sum_loss1 +=
          __shfl_xor_sync(0xffffffff, sum_loss1, mask,
                          warpSize);  // WARP_SHFL_XOR(sum_loss1, mask);
      sum_loss2 +=
          __shfl_xor_sync(0xffffffff, sum_loss2, mask,
                          warpSize);  // WARP_SHFL_XOR(sum_loss2, mask);
    }
    // inter-warp reductions
    if (BDIMY > 1) {
      __shared__ U buf[BDIMX * BDIMY];
      for (int offset = BDIMY / 2; offset > 0; offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2 * offset) {
          const int wrt_i = (threadIdx.y - offset) * BDIMX + threadIdx.x;
          buf[2 * wrt_i] = sum_loss1;
          buf[2 * wrt_i + 1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          sum_loss1 += buf[2 * read_i];
          sum_loss2 += buf[2 * read_i + 1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        buf[2 * threadIdx.x] = sum_loss1;
        buf[2 * threadIdx.x + 1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y != 0) {
        sum_loss1 = buf[2 * threadIdx.x];
        sum_loss2 = buf[2 * threadIdx.x + 1];
      }
    }
    // all threads now have the two sums over l
    U fH = (U)n2;
    U term1 = (U(1) / fH) * c_invvar;
    T *k_grad_input = grad_input + i1 * n2;
    if (gamma != NULL) {
      for (int l = thrx; l < n2; l += numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss * gamma[l];
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx; l < n2; l += numx) {
        const U c_h = static_cast<U>(k_input[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss;
        f_grad_input -= sum_loss1;
        f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
  }
}

// Make sure that d_scale != nullptr && d_bias != nullptr
// Since d_scale != nullptr, scale would not be nullptr
template <typename T, typename U, int BlockDim, bool HasDx>
__global__ void LayerNormBackwardGradientAll(const T *x, const T *d_y,
                                             U *d_scale, U *d_bias, T *d_x,
                                             const U *mean, const U *var,
                                             const U *scale, float epsilon,
                                             int batch_size, int feature_size,
                                             int col_offset) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int beg_idx = threadIdx.x * feature_size + (blockIdx.x + col_offset);
  int end_idx = batch_size * feature_size + (blockIdx.x + col_offset);
  int stride = BlockDim * feature_size;

  U d_scale_partial = static_cast<U>(0), d_bias_partial = static_cast<U>(0);

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val = real_sqrt(static_cast<U>(var[row_idx]) + epsilon);
    d_scale_partial += static_cast<U>(d_y[i]) *
                       (static_cast<U>(x[i]) - mean[row_idx]) / var_val;
    d_bias_partial += static_cast<U>(d_y[i]);
    if (HasDx) {
      d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) *
                              scale[blockIdx.x + col_offset] / var_val);
    }
  }

  auto pair = BlockReduce(temp_storage)
                  .Reduce(PairForLayerNorm<U>(d_scale_partial, d_bias_partial),
                          PairForLayerNormAddFunctor<U>());

  if (threadIdx.x == 0) {
    d_scale[blockIdx.x + col_offset] = pair.first_;
    d_bias[blockIdx.x + col_offset] = pair.second_;
  }
}

// Make sure that there is only one true expression: d_scale != nullptr
// or d_bias != nullptr
// Notice: scale may be nullptr
template <typename T, typename U, int BlockDim, bool HasDx, bool HasDScale>
__global__ void LayerNormBackwardGradientScaleOrBias(
    const T *x, const T *d_y, U *d_scale, U *d_bias, T *d_x, const U *mean,
    const U *var, const U *scale, float epsilon, int batch_size,
    int feature_size, int col_offset) {
  using BlockReduce = hipcub::BlockReduce<U, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int beg_idx = threadIdx.x * feature_size + blockIdx.x + col_offset;
  int end_idx = batch_size * feature_size + blockIdx.x + col_offset;
  int stride = BlockDim * feature_size;
  U d_scale_or_d_bias_partial = static_cast<U>(0);

  for (int i = beg_idx; i < end_idx; i += stride) {
    int row_idx = i / feature_size;
    auto var_val =
        static_cast<U>(real_sqrt(static_cast<float>(var[row_idx]) + epsilon));
    if (HasDScale) {
      d_scale_or_d_bias_partial += static_cast<U>(d_y[i]) *
                                   (static_cast<U>(x[i]) - mean[row_idx]) /
                                   var_val;
    } else {  // d_bias != nullptr
      d_scale_or_d_bias_partial += static_cast<U>(d_y[i]);
    }

    if (HasDx) {
      if (scale != nullptr) {
        d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) *
                                scale[blockIdx.x + col_offset] / var_val);
      } else {
        d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) / var_val);
      }
    }
  }

  d_scale_or_d_bias_partial =
      BlockReduce(temp_storage).Reduce(d_scale_or_d_bias_partial, hipcub::Sum());

  if (threadIdx.x == 0) {
    if (HasDScale) {
      d_scale[blockIdx.x + col_offset] = d_scale_or_d_bias_partial;
    } else {
      d_bias[blockIdx.x + col_offset] = d_scale_or_d_bias_partial;
    }
  }
}

template <typename T, typename U, int BlockDim>
__global__ void LayerNormBackwardPostProcessToCalculateDX(const T *x, T *d_x,
                                                          const U *mean,
                                                          const U *var,
                                                          float epsilon,
                                                          int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ U d_x_reduce_tmp[2];

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  U block_mean = mean[blockIdx.x];
  U block_var = var[blockIdx.x];
  U d_x_mean_partial = static_cast<U>(0), d_x_var_partial = static_cast<U>(0);
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x_mean_partial += static_cast<U>(d_x[i]);
    d_x_var_partial +=
        static_cast<U>(d_x[i]) * (static_cast<U>(x[i]) - block_mean);
  }

  auto pair =
      BlockReduce(temp_storage)
          .Reduce(PairForLayerNorm<U>(d_x_mean_partial, d_x_var_partial),
                  PairForLayerNormAddFunctor<U>());

  if (threadIdx.x == 0) {
    d_x_reduce_tmp[0] = static_cast<float>(pair.first_) / feature_size;
    d_x_reduce_tmp[1] =
        static_cast<float>(pair.second_) /
        (feature_size * (static_cast<float>(block_var) + epsilon));
  }
  __syncthreads();

  d_x_mean_partial = d_x_reduce_tmp[0];
  d_x_var_partial = d_x_reduce_tmp[1];
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x[i] -= static_cast<T>(d_x_mean_partial);
    d_x[i] -=
        static_cast<T>((static_cast<U>(x[i]) - block_mean) * d_x_var_partial);
  }
}

// Here, we only calculate d_x
template <typename T, typename U, int BlockDim>
__global__ void LayerNormBackwardGradientOnlyDX(const T *x, const T *d_y,
                                                T *d_x, const U *mean,
                                                const U *var, const U *scale,
                                                float epsilon,
                                                int feature_size) {
  using BlockReduce = hipcub::BlockReduce<PairForLayerNorm<U>, BlockDim>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ U d_x_reduce_tmp[2];

  int beg_idx = blockIdx.x * feature_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * feature_size;

  U block_mean = mean[blockIdx.x], block_var = var[blockIdx.x];
  U d_x_mean_partial = static_cast<U>(0), d_x_var_partial = static_cast<U>(0);
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    auto var_val =
        static_cast<U>(real_sqrt(static_cast<float>(block_var) + epsilon));
    if (scale != nullptr) {
      int col_idx = i % feature_size;
      d_x[i] =
          static_cast<T>(static_cast<U>(d_y[i]) * scale[col_idx] / var_val);
    } else {
      d_x[i] = static_cast<T>(static_cast<U>(d_y[i]) / var_val);
    }
    d_x_mean_partial += static_cast<U>(d_x[i]);
    d_x_var_partial +=
        static_cast<U>(d_x[i]) * (static_cast<U>(x[i]) - block_mean);
  }

  auto pair =
      BlockReduce(temp_storage)
          .Reduce(PairForLayerNorm<U>(d_x_mean_partial, d_x_var_partial),
                  PairForLayerNormAddFunctor<U>());

  if (threadIdx.x == 0) {
    d_x_reduce_tmp[0] = static_cast<float>(pair.first_) / feature_size;
    d_x_reduce_tmp[1] =
        static_cast<float>(pair.second_) /
        (feature_size * (static_cast<float>(block_var) + epsilon));
  }
  __syncthreads();

  d_x_mean_partial = d_x_reduce_tmp[0];
  d_x_var_partial = d_x_reduce_tmp[1];
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    d_x[i] -= static_cast<T>(d_x_mean_partial);
    d_x[i] -=
        static_cast<T>((static_cast<U>(x[i]) - block_mean) * d_x_var_partial);
  }
}

template <typename T, typename U>
__global__ void LayerNormBackwardWhenBatchSizeIsOne(
    const T *x, const T *d_y, T *d_x, U *d_scale, U *d_bias, const U *mean,
    const U *var, const U *scale, float epsilon, int feature_size) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < feature_size) {
    auto var_val =
        static_cast<U>(real_sqrt(static_cast<float>(var[idx]) + epsilon));
    if (d_x != nullptr) {
      if (d_scale == nullptr) {
        d_x[idx] = static_cast<T>(static_cast<U>(d_y[idx]) / var_val);
      } else {
        d_x[idx] =
            static_cast<T>(static_cast<U>(d_y[idx]) * scale[idx] / var_val);
      }
    }

    if (d_scale != nullptr) {
      d_scale[idx] = static_cast<U>(d_y[idx]) *
                     (static_cast<U>(x[idx]) - mean[idx]) / var_val;
    }

    if (d_bias != nullptr) d_bias[idx] = static_cast<U>(d_y[idx]);
  }
}

template <typename T, typename U>
static void LayerNormBackward(const T *x, const T *d_y, const U *scale,
                              const U *mean, const U *var, T *d_x, U *d_scale,
                              U *d_bias, float epsilon, int batch_size,
                              int feature_size,
                              const framework::ExecutionContext &ctx) {
  auto &dev_ctx = ctx.cuda_device_context();
  auto stream = dev_ctx.stream();

  const int kMaxBlockDim = 512;
  const int kMaxBlockNum = 128;
  int gradient_flag = ((d_x != nullptr ? 1 : 0) << 2) |
                      ((d_scale != nullptr ? 1 : 0) << 1) |
                      ((d_bias != nullptr ? 1 : 0));
  if (gradient_flag == 0) return;

  if (batch_size == 1) {
    LayerNormBackwardWhenBatchSizeIsOne<
        T, U><<<(feature_size + kMaxBlockDim - 1) / kMaxBlockDim, kMaxBlockDim,
                0, stream>>>(x, d_y, d_x, d_scale, d_bias, mean, var, scale,
                             epsilon, feature_size);

    if (d_x != nullptr) {
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(LayerNormBackwardPostProcessToCalculateDX<
                             T, U, kBlockDim><<<1, kBlockDim, 0, stream>>>(
            x, d_x, mean, var, epsilon, feature_size));
      }
    }
    return;
  }

  auto block_dim = GetDesiredBlockDim(batch_size);
  switch (gradient_flag) {
    case 1:  // d_x == nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, false,
                false><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      break;
    case 2:  // d_x == nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, false,
                true><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      break;
    case 3:  // d_x == nullptr, d_scale != nulptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientAll<
                T, U, kBlockDim, false><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      break;
    case 4:  // d_x != nullptr, d_scale == nullptr, d_bias == nullptr
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardGradientOnlyDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_y, d_x, mean, var, scale, epsilon, feature_size));
      }
      break;
    case 5:  // d_x != nulptr, d_scale == nullptr, d_bias != nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, true,
                false><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    case 6:  // d_x != nullptr, d_scale != nullptr, d_bias == nullptr
      switch (block_dim) {
        FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE(
            feature_size, kMaxBlockNum,
            LayerNormBackwardGradientScaleOrBias<
                T, U, kBlockDim, true,
                true><<<block_num, kBlockDim, 0, stream>>>(
                x, d_y, d_scale, d_bias, d_x, mean, var, scale, epsilon,
                batch_size, feature_size, col_offset));
      }
      switch (GetDesiredBlockDim(feature_size)) {
        FIXED_BLOCK_DIM_CASE(
            LayerNormBackwardPostProcessToCalculateDX<
                T, U, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
                x, d_x, mean, var, epsilon, feature_size));
      }
      break;
    case 7:  // d_x != nullptr, d_scale != nullptr, d_bias != nullptr
    {
      constexpr int VPT = 4;
      constexpr int BDIMX2 = 32;
      constexpr int BDIMY2 = 4;
      dim3 threads2(BDIMX2, BDIMY2, 1);
      constexpr int part_size = BDIMY2 * VPT;
      const dim3 blocks2((feature_size + BDIMX2 - 1) / BDIMX2, part_size, 1);

      auto part_grad_gamma_ptr =
          memory::Alloc(dev_ctx, part_size * feature_size * sizeof(U));
      auto part_grad_beta_ptr =
          memory::Alloc(dev_ctx, part_size * feature_size * sizeof(U));
      U *part_grad_gamma = reinterpret_cast<U *>(part_grad_gamma_ptr->ptr());
      U *part_grad_beta = reinterpret_cast<U *>(part_grad_beta_ptr->ptr());

      LayerNormBackwardPartGradGammaBeta<T, U, BDIMX2, BDIMY2,
                                         VPT><<<blocks2, threads2, 0, stream>>>(
          d_y, x, batch_size, feature_size, mean, var, epsilon, part_grad_gamma,
          part_grad_beta);  // compute part_grad_gamma, beta

      constexpr int BDIMX3 = 32;
      constexpr int BDIMY3 = 8;
      dim3 threads3(BDIMX3, BDIMY3, 1);
      const dim3 blocks3((feature_size + BDIMX2 - 1) / BDIMX2, 1, 1);
      LayerNormBackwardSumGradGammaBeta<
          T, U, BDIMX3, BDIMY3><<<blocks3, threads3, 0, stream>>>(
          part_grad_gamma, part_grad_beta, part_size, batch_size, feature_size,
          d_scale, d_bias);

      constexpr int BDIMX1 = 32;
      constexpr int BDIMY1 = 4;
      dim3 threads1(BDIMX1, BDIMY1, 1);
      const dim3 blocks1(1, batch_size, 1);
      LayerNormBackwardComputeGradInput<
          T, U, BDIMX1, BDIMY1><<<blocks1, threads1, 0, stream>>>(
          d_y, x, batch_size, feature_size, mean, var, epsilon, scale, d_x);
      break;
    }
    default:
      break;
  }
}

template <typename T>
void LayerNormDirectCUDAFunctor<T>::operator()(hipStream_t stream,
                                               const T *input,
                                               std::vector<int> input_shape,
                                               const T *bias, const T *scale,
                                               T *output, T *mean, T *variance,
                                               int begin_norm_axis, float eps) {
  const auto x_dims = framework::make_ddim(input_shape);
  auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
  int batch_size = static_cast<int>(matrix_dim[0]);
  int feature_size = static_cast<int>(matrix_dim[1]);
  switch (GetDesiredBlockDim(feature_size)) {
    FIXED_BLOCK_DIM_CASE(
        LayerNormForward<T, T, kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
            input, scale, bias, output, mean, variance, eps, feature_size));
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Product from begin_norm_axis to end in layer_norm must be larger "
          "than 1"));
      break;
  }
}

template <typename T>
class LayerNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *bias = ctx.Input<Tensor>("Bias");
    auto *x = ctx.Input<Tensor>("X");

    auto *y = ctx.Output<Tensor>("Y");
    auto *mean = ctx.Output<Tensor>("Mean");
    auto *var = ctx.Output<Tensor>("Variance");
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");

    const auto x_dims = x->dims();
    auto *x_data = x->data<T>();
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());
    auto *mean_data = mean->mutable_data<U>(ctx.GetPlace());
    auto *var_data = var->mutable_data<U>(ctx.GetPlace());
    auto *scale_data = (scale == nullptr ? nullptr : scale->data<U>());
    auto *bias_data = (bias == nullptr ? nullptr : bias->data<U>());

    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    auto stream = ctx.cuda_device_context().stream();

    switch (GetDesiredBlockDim(feature_size)) {
      FIXED_BLOCK_DIM_CASE(
          LayerNormForward<T, U,
                           kBlockDim><<<batch_size, kBlockDim, 0, stream>>>(
              x_data, scale_data, bias_data, y_data, mean_data, var_data,
              epsilon, feature_size));
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "Product from begin_norm_axis to end must be larger than 1"));
        break;
    }
  }
};

template <typename T>
class LayerNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const float epsilon = ctx.Attr<float>("epsilon");
    // d_x, d_scale, d_bias may be nullptr
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *x = ctx.Input<Tensor>("X");
    auto *mean = ctx.Input<Tensor>("Mean");
    auto *var = ctx.Input<Tensor>("Variance");
    auto *scale = ctx.Input<Tensor>("Scale");
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    auto *x_data = x->data<T>();
    auto *d_y_data = d_y->data<T>();
    auto *mean_data = mean->data<U>();
    auto *var_data = var->data<U>();

    auto *scale_data = (scale == nullptr ? nullptr : scale->data<U>());
    auto *d_scale_data =
        (d_scale == nullptr ? nullptr
                            : d_scale->mutable_data<U>(ctx.GetPlace()));
    auto *d_bias_data =
        (d_bias == nullptr ? nullptr : d_bias->mutable_data<U>(ctx.GetPlace()));
    auto *d_x_data =
        (d_x == nullptr ? nullptr : d_x->mutable_data<T>(ctx.GetPlace()));

    const auto &x_dims = x->dims();
    const auto begin_norm_axis = ctx.Attr<int>("begin_norm_axis");
    auto matrix_dim = framework::flatten_to_2d(x_dims, begin_norm_axis);
    int batch_size = static_cast<int>(matrix_dim[0]);
    int feature_size = static_cast<int>(matrix_dim[1]);

    LayerNormBackward<T, U>(x_data, d_y_data, scale_data, mean_data, var_data,
                            d_x_data, d_scale_data, d_bias_data, epsilon,
                            batch_size, feature_size, ctx);
  }
};

template class LayerNormDirectCUDAFunctor<float>;

#undef FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE_BASE
#undef FIXED_BLOCK_DIM_FIXED_BLOCK_NUM_CASE
#undef FIXED_BLOCK_DIM_CASE_BASE
#undef FIXED_BLOCK_DIM_CASE
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    layer_norm,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    layer_norm_grad,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LayerNormGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>);
