#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <vector>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__device__ T dmcn_get_gradient_weight(T argmax_h, T argmax_w, const int h,
                                      const int w, const int height,
                                      const int width) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  T weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename T>
__global__ void modulated_deformable_col2im_gpu_kernel(
    const int nthreads, const T* data_col, const T* data_offset,
    const T* data_mask, const int channels, const int height, const int width,
    const int kernel_h, const int kernel_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int dilation_h,
    const int dilation_w, const int channel_per_deformable_group,
    const int batch_size, const int deformable_group, const int height_col,
    const int width_col, T* grad_im) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t thread = index; thread < nthreads; thread += offset) {
    const int j = (thread / width_col / height_col / batch_size) % kernel_w;
    const int i =
        (thread / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c =
        thread / width_col / height_col / batch_size / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = thread % width_col;
    int h_out = (thread / width_col) % height_col;
    int b = (thread / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const T* data_offset_ptr = data_offset +
                               (b * deformable_group + deformable_group_index) *
                                   2 * kernel_h * kernel_w * height_col *
                                   width_col;
    const T* data_mask_ptr = data_mask +
                             (b * deformable_group + deformable_group_index) *
                                 kernel_h * kernel_w * height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const int data_mask_hw_ptr =
        ((i * kernel_w + j) * height_col + h_out) * width_col + w_out;
    const T offset_h = data_offset_ptr[data_offset_h_ptr];
    const T offset_w = data_offset_ptr[data_offset_w_ptr];
    const T mask = data_mask_ptr[data_mask_hw_ptr];
    const T cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const T cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const T cur_top_grad = data_col[thread] * mask;
    const int cur_h = static_cast<int>(cur_inv_h_data);
    const int cur_w = static_cast<int>()cur_inv_w_data);
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          T weight =
              dmcn_get_gradient_weight(cur_inv_h_data, cur_inv_w_data,
                                       cur_h + dy, cur_w + dx, height, width);

          atomicAdd(&grad_im[cur_bottom_grad_pos], weight * cur_top_grad);
        }
      }
    }
  }
}

template <typename T>
inline void modulated_deformable_col2im(
    // const paddle::platform::CUDADeviceContext ctx,
    const platform::DeviceContext& ctx, const T* data_col, const T* data_offset,
    const T* data_mask, const std::vector<int64_t> im_shape,
    const std::vector<int64_t> col_shape,
    const std::vector<int64_t> kernel_shape, const std::vector<int> pad,
    const std::vector<int> stride, const std::vector<int> dilation,
    const int deformable_group, T* grad_im) {
  int channel_per_deformable_group = im_shape[0] / deformable_group;
  int num_kernels = col_shape[0] * col_shape[1] * col_shape[2] * col_shape[3];
  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  modulated_deformable_col2im_gpu_kernel<T><<<
      blocks, threads, 0,
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
      num_kernels, data_col, data_offset, data_mask, im_shape[0], im_shape[1],
      im_shape[2], kernel_shape[2], kernel_shape[3], pad[0], pad[1], stride[0],
      stride[1], dilation[0], dilation[1], channel_per_deformable_group,
      col_shape[1], deformable_group, col_shape[2], col_shape[3], grad_im);
}

template <typename T>
__device__ T dmcn_get_coordinate_weight(T argmax_h, T argmax_w,
                                        const int height, const int width,
                                        const T* im_data, const int data_width,
                                        const int bp_dir) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  T weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) *
                im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) *
                im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) *
                im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) *
                im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) *
                im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) *
                im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) *
                im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) *
                im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename T>
__global__ void deforamble_col2im_coord_gpu_kernel(
    const int nthreads, const T* data_col, const T* data_im,
    const T* data_offset, const T* data_mask, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int batch_size,
    const int offset_channels, const int deformable_group, const int height_col,
    const int width_col, T* grad_offset, T* grad_mask) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    T val = 0, mval = 0;
    const int w = nthreads % width_col;
    const int h = (nthreads / width_col) % height_col;
    const int c = (nthreads / width_col / height_col) % offset_channels;
    const int b = (nthreads / width_col / height_col) / offset_channels;

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const T* data_col_ptr = data_col +
                            deformable_group_index *
                                channel_per_deformable_group * batch_size *
                                width_col * height_col;
    const T* data_im_ptr = data_im +
                           (b * deformable_group + deformable_group_index) *
                               channel_per_deformable_group / kernel_h /
                               kernel_w * height * width;
    const T* data_offset_ptr = data_offset +
                               (b * deformable_group + deformable_group_index) *
                                   2 * kernel_h * kernel_w * height_col *
                                   width_col;
    const T* data_mask_ptr = data_mask +
                             (b * deformable_group + deformable_group_index) *
                                 kernel_h * kernel_w * height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = offset_c / 2; col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos =
          (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i =
          (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const int data_mask_hw_ptr =
          (((i * kernel_w + j) * height_col + h_out) * width_col + w_out);
      const T offset_h = data_offset_ptr[data_offset_h_ptr];
      const T offset_w = data_offset_ptr[data_offset_w_ptr];
      const T mask = data_mask_ptr[data_mask_hw_ptr];
      T inv_h = h_in + i * dilation_h + offset_h;
      T inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -2;
      } else {
        mval += data_col_ptr[col_pos] *
                dmcn_im2col_bilinear(data_im_ptr + cnt * height * width, width,
                                     height, width, inv_h, inv_w);
      }
      const T weight = dmcn_get_coordinate_weight(
          inv_h, inv_w, height, width, data_im_ptr + cnt * height * width,
          width, bp_dir);
      val += weight * data_col_ptr[col_pos] * mask;
      cnt += 1;
    }
    grad_offset[i] = val;
    if (offset_c % 2 == 0)
      grad_mask[(((b * deformable_group + deformable_group_index) * kernel_h *
                      kernel_w +
                  offset_c / 2) *
                     height_col +
                 h) *
                    width_col +
                w] = mval;
  }
}

template <typename T>
inline void modulated_deformable_col2im_coord(
    // const paddle::platform::CUDADeviceContext ctx,
    const platform::DeviceContext& ctx, const T* data_col, const T* data_im,
    const T* data_offset, const T* data_mask,
    const std::vector<int64_t> im_shape, const std::vector<int64_t> col_shape,
    const std::vector<int64_t> kernel_shape, const std::vector<int> paddings,
    const std::vector<int> strides, const std::vector<int> dilations,
    const int deformable_groups, T* grad_offset, T* grad_mask) {
  int num_kernels = 2 * kernel_shape[2] * kernel_shape[3] * col_shape[1] *
                    col_shape[2] * col_shape[3] * deformable_groups;
  int channel_per_deformable_group = col_shape[0] / deformable_groups;
  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  deforamble_col2im_coord_gpu_kernel<T><<<
      blocks, threads, 0,
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
      num_kernels, data_col, data_im, data_offset, data_mask, im_shape[0],
      im_shape[1], im_shape[2], kernel_shape[2], kernel_shape[3], paddings[0],
      paddings[1], strides[0], strides[1], dilations[0], dilations[1],
      channel_per_deformable_group, col_shape[1],
      2 * kernel_shape[2] * kernel_shape[3] * deformable_groups,
      deformable_groups, col_shape[2], col_shape[3], grad_offset, grad_mask);
}

template <typename T>
__device__ T dmcn_im2col_bilinear(const T* bottom_data, const int data_width,
                                  const int height, const int width, T h, T w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  T lh = h - h_low;
  T lw = w - w_low;
  T hh = 1 - lh, hw = 1 - lw;

  T v1 = 0;
  if (h_low >= 0 && w_low >= 0) v1 = bottom_data[h_low * data_width + w_low];
  T v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  T v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  T v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  T w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename T>
__global__ void modulated_deformable_im2col_gpu_kernel(
    const int nthreads, const T* data_im, const T* data_offset,
    const T* data_mask, const int height, const int width, const int kernel_h,
    const int kernel_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int batch_size,
    const int num_channels, const int deformable_group, const int height_col,
    const int width_col, T* data_col) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    // index of output matrix
    const int w_col = i % width_col;
    const int h_col = (i / width_col) % height_col;
    const int b_col = (i / width_col) / height_col % batch_size;
    const int c_im = (i / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // conpute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    T* data_col_ptr =
        data_col +
        ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    const T* data_im_ptr =
        data_im + (b_col * num_channels + c_im) * height * width;
    const T* data_offset_ptr =
        data_offset +
        (b_col * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;
    const T* data_mask_ptr =
        data_mask +
        (b_col * deformable_group + deformable_group_index) * kernel_h *
            kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;

        const T offset_h = data_offset_ptr[data_offset_h_ptr];
        const T offset_w = data_offset_ptr[data_offset_w_ptr];
        const T mask = data_mask_ptr[data_mask_hw_ptr];
        T val = static_cast<T>(0);
        const T h_im = h_in + i * dilation_h + offset_h;
        const T w_im = w_in + j * dilation_w + offset_w;

        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
          val = dmcn_im2col_bilinear(data_im_ptr, width, height, width, h_im,
                                     w_im);
        }
        *data_col_ptr = val * mask;
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}

// im_shape {c_i, i_h, i_w}
// col_shape {c_in * k_h * k_w, im2col_step, o_h, o_w}
// filter_shape {c_o, c_i, k_h, k_w}
// paddings {p_h, p_w}
// strides {s_h, s_w}
// dilations {d_h, d_w}
template <typename T>
inline void modulated_deformable_im2col(
    // const paddle::platform::CUDADeviceContext ctx,
    const platform::DeviceContext& ctx, const T* data_im, const T* data_offset,
    const T* data_mask, const std::vector<int64_t> im_shape,
    const std::vector<int64_t> col_shape,
    const std::vector<int64_t> filter_shape, const std::vector<int> paddings,
    const std::vector<int> strides, const std::vector<int> dilations,
    const int deformable_groups, T* data_col) {
  // {c_i / deformable_group}
  int channel_per_deformable_group = im_shape[0] / deformable_groups;
  // {c_i * o_h * o_w}
  int num_kernels = im_shape[0] * col_shape[1] * col_shape[2] * col_shape[3];

  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  modulated_deformable_im2col_gpu_kernel<T><<<
      blocks, threads, 0,
      reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
      num_kernels, data_im, data_offset, data_mask, im_shape[1], im_shape[2],
      filter_shape[2], filter_shape[3], paddings[0], paddings[1], strides[0],
      strides[1], dilations[0], dilations[1], channel_per_deformable_group,
      col_shape[1], im_shape[0], deformable_groups, col_shape[2], col_shape[3],
      data_col);
}

template <typename DeviceContext, typename T>
class ModulatedDeformableConvCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const Tensor* input = ctx.Input<Tensor>("Input");
    const Tensor offset = *ctx.Input<Tensor>("Offset");
    const Tensor mask = *ctx.Input<Tensor>("Mask");
    Tensor filter = *ctx.Input<Tensor>("Filter");
    Tensor* output = ctx.Output<Tensor>("Output");
    output->mutable_data<T>(ctx.GetPlace());

    const int groups = ctx.Attr<int>("groups");
    const int deformable_groups = ctx.Attr<int>("deformable_groups");
    const int im2col_step = ctx.Attr<int>("im2col_step");
    const std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    const std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    const std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");

    auto& dev_ctx = ctx.cuda_device_context();

    const int batch_size = static_cast<int>(input->dims()[0]);

    // filter_shape_vec: {c_o, c_i, k_h, k_w}
    std::vector<int64_t> filter_shape_vec(framework::vectorize(filter.dims()));
    // output_shape_vec: {n, o_c, o_h, o_w}
    std::vector<int64_t> output_shape_vec(framework::vectorize(output->dims()));

    // filter_shape_vec.size(): 4
    // col_shape_vec: {c_i * k_h * k_w, im2col_step, o_h, o_w}
    size_t data_dim = filter_shape_vec.size() - 2;
    std::vector<int64_t> col_buffer_shape_vec(2 + data_dim);
    // c_i * k_w * k_h /
    col_buffer_shape_vec[0] =
        input->dims()[1] * filter.dims()[2] * filter.dims()[3];
    col_buffer_shape_vec[1] = im2col_step;
    for (size_t j = 0; j < data_dim; ++j) {
      col_buffer_shape_vec[j + 2] = output_shape_vec[j + 2];
    }
    framework::DDim col_shape(framework::make_ddim(col_buffer_shape_vec));
    std::vector<int64_t> output_buffer_shape_vec(1);
    output_buffer_shape_vec[0] = batch_size * output_shape_vec[1] *
                                 output_shape_vec[2] * output_shape_vec[3];
    framework::DDim output_shape(framework::make_ddim(output_buffer_shape_vec));
    Tensor col_buffer;
    Tensor output_buffer;
    col_buffer = ctx.AllocateTmpTensor<T, DeviceContext>(col_shape, dev_ctx);
    output_buffer =
        ctx.AllocateTmpTensor<T, DeviceContext>(output_shape, dev_ctx);

    int64_t M = output_shape_vec[1] / groups;
    int64_t N = im2col_step * output_shape_vec[2] * output_shape_vec[3];
    int64_t K =
        input->dims()[1] * filter_shape_vec[2] * filter_shape_vec[3] / groups;

    Tensor weight_3d;
    weight_3d.ShareDataWith(filter);
    weight_3d.Resize(framework::make_ddim({groups, M, K}));
    Tensor col_buffer_3d;
    col_buffer_3d.ShareDataWith(col_buffer);
    col_buffer_3d.Resize(framework::make_ddim({groups, K, N}));
    Tensor output_4d;
    output_4d.ShareDataWith(output_buffer);
    output_4d.Resize(
        framework::make_ddim({batch_size / im2col_step, groups, M, N}));

    // // input {c_i, i_h, i_w}
    framework::DDim input_shape =
        framework::slice_ddim(input->dims(), 1, input->dims().size());
    std::vector<int64_t> input_shape_vec = framework::vectorize(input_shape);

    int input_dim = input->numel() / input->dims()[0];
    int input_offset_dim = offset.numel() / offset.dims()[0];
    int input_mask_dim = mask.numel() / mask.dims()[0];

    auto blas = math::GetBlas<DeviceContext, T>(dev_ctx);

    for (int i = 0; i < batch_size / im2col_step; i++) {
      modulated_deformable_im2col(
          ctx.device_context(), input->data<T>() + i * im2col_step * input_dim,
          offset.data<T>() + i * im2col_step * input_offset_dim,
          mask.data<T>() + i * im2col_step * input_mask_dim, input_shape_vec,
          col_buffer_shape_vec, filter_shape_vec, paddings, strides, dilations,
          deformable_groups, col_buffer.mutable_data<T>(ctx.GetPlace()));

      Tensor output_3d = output_4d.Slice(i, i + 1).Resize(
          framework::slice_ddim(output_4d.dims(), 1, output_4d.dims().size()));
      for (int g = 0; g < groups; g++) {
        Tensor weight_3d_slice =
            weight_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                weight_3d.dims(), 1, weight_3d.dims().size()));
        Tensor col_buffer_3d_slice =
            col_buffer_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                col_buffer_3d.dims(), 1, col_buffer_3d.dims().size()));
        Tensor output_3d_slice =
            output_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                output_3d.dims(), 1, output_3d.dims().size()));
        // gemm
        blas.MatMul(weight_3d_slice, false, col_buffer_3d_slice, false, T(1.0),
                    &output_3d_slice, T(0.0));
      }
    }
    Tensor trans_output_4d;
    trans_output_4d.ShareDataWith(output_buffer);
    framework::DDim trans_output_4d_shape = {
        batch_size / im2col_step, filter_shape_vec[0], im2col_step,
        output_shape_vec[2] * output_shape_vec[3]};
    trans_output_4d.Resize(trans_output_4d_shape);

    Tensor origin_output_4d;
    origin_output_4d.ShareDataWith(*output);
    framework::DDim origin_output_4d_shape = {
        batch_size / im2col_step, im2col_step, filter_shape_vec[0],
        output_shape_vec[2] * output_shape_vec[3]};
    // swap axis
    origin_output_4d = trans_output_4d.Resize(origin_output_4d_shape);
    // TODO(yifan): check bias
  }
};

template <typename DeviceContext, typename T>
class ModulatedDeformableConvGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const Tensor* output_grad =
        ctx.Input<Tensor>(framework::GradVarName("Output"));
    Tensor* input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    Tensor* filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));
    Tensor* offset_grad = ctx.Output<Tensor>(framework::GradVarName("Offset"));
    Tensor* mask_grad = ctx.Output<Tensor>(framework::GradVarName("mask"));

    const Tensor* input = ctx.Input<Tensor>("Input");
    Tensor offset = *ctx.Input<Tensor>("Offset");
    Tensor mask = *ctx.Input<Tensor>("Mask");
    Tensor filter = *ctx.Input<Tensor>("Filter");

    if (!input_grad && !filter_grad && !offset_grad && !mask_grad) return;

    int groups = ctx.Attr<int>("groups");
    int deformable_groups = ctx.Attr<int>("deformable_groups");
    int im2col_step = ctx.Attr<int>("im2col_step");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");

    auto& dev_ctx = ctx.cuda_device_context();
    const int batch_size = static_cast<int>(input->dims()[0]);

    framework::DDim input_shape =
        framework::slice_ddim(input->dims(), 1, input->dims().size());
    std::vector<int64_t> input_shape_vec = framework::vectorize(input_shape);

    // filter_shape_vec: {c_o, c_i, k_h, k_w}
    std::vector<int64_t> filter_shape_vec(framework::vectorize(filter.dims()));
    // output_shape_vec: {n, o_c, o_h, o_w}
    std::vector<int64_t> output_shape_vec(
        framework::vectorize(output_grad->dims()));

    // get col_shape in the im2col calculation
    size_t data_dim = filter_shape_vec.size() - 2;
    // col_buffer_shape_vec {c_i * k_h * k_w, im2col_step, o_h, o_w}
    std::vector<int64_t> col_buffer_shape_vec(data_dim + 2);
    col_buffer_shape_vec[0] =
        input->dims()[1] * filter.dims()[2] * filter.dims()[3];
    col_buffer_shape_vec[1] = im2col_step;
    for (size_t j = 0; j < data_dim; ++j) {
      col_buffer_shape_vec[j + 2] = output_shape_vec[j + 2];
    }
    framework::DDim col_shape(framework::make_ddim(col_buffer_shape_vec));
    std::vector<int64_t> output_buffer_shape_vec(1);
    output_buffer_shape_vec[0] = batch_size * output_shape_vec[1] *
                                 output_shape_vec[2] * output_shape_vec[3];
    framework::DDim output_shape(framework::make_ddim(output_buffer_shape_vec));
    Tensor col_buffer;
    Tensor output_buffer;
    col_buffer = ctx.AllocateTmpTensor<T, DeviceContext>(col_shape, dev_ctx);
    output_buffer =
        ctx.AllocateTmpTensor<T, DeviceContext>(output_shape, dev_ctx);

    Tensor trans_output_4d;
    framework::DDim trans_output_4d_shape = {
        batch_size / im2col_step, filter_shape_vec[0], im2col_step,
        output_shape_vec[2] * output_shape_vec[3]};
    trans_output_4d.ShareDataWith(output_buffer);
    trans_output_4d.Resize(trans_output_4d_shape);

    Tensor origin_output_4d;
    framework::DDim origin_output_4d_shape = {
        batch_size / im2col_step, im2col_step, filter_shape_vec[0],
        output_shape_vec[2] * output_shape_vec[3]};
    origin_output_4d.ShareDataWith(*output_grad);
    trans_output_4d = origin_output_4d.Resize(trans_output_4d_shape);

    int64_t M = input_shape_vec[0] / groups;
    int64_t N = im2col_step * output_shape_vec[2] * output_shape_vec[3];
    int64_t K = filter_shape_vec[1] * filter_shape_vec[2] *
                filter_shape_vec[3] / groups;

    framework::DDim weight_3d_shape = {groups, K, M};
    framework::DDim out_grad_4d_shape = {batch_size / im2col_step, groups, K,
                                         N};
    framework::DDim col_buffer_3d_shape = {groups, M, N};
    framework::DDim dweight_3d_shape = {groups, K, M};
    framework::DDim data_grad_shape = {input_grad->numel()};

    Tensor weight_3d;
    weight_3d.ShareDataWith(filter);
    weight_3d.Resize(weight_3d_shape);
    Tensor out_grad_4d;
    out_grad_4d.ShareDataWith(output_buffer);
    out_grad_4d.Resize(out_grad_4d_shape);
    Tensor col_buffer_3d;
    col_buffer_3d.ShareDataWith(col_buffer);
    col_buffer_3d.Resize(col_buffer_3d_shape);
    Tensor dweight_3d;
    dweight_3d.ShareDataWith(*filter_grad);
    dweight_3d.Resize(dweight_3d_shape);
    Tensor data_grad;
    data_grad.ShareDataWith(*input_grad);
    data_grad.Resize(data_grad_shape);

    math::SetConstant<DeviceContext, T> set_zero;
    auto blas = math::GetBlas<DeviceContext, T>(dev_ctx);

    set_zero(dev_ctx, &data_grad, static_cast<T>(0));

    int input_dim = input->numel() / input->dims()[0];
    int input_offset_dim = offset.numel() / offset.dims()[0];
    int input_mask_dim = mask.numel() / mask.dims()[0];

    for (int i = 0; i < batch_size / im2col_step; i++) {
      Tensor out_grad_3d =
          out_grad_4d.Slice(i, i + 1).Resize(framework::slice_ddim(
              out_grad_4d.dims(), 1, out_grad_4d.dims().size()));
      for (int g = 0; g < groups; g++) {
        Tensor weight_3d_slice =
            weight_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                weight_3d.dims(), 1, weight_3d.dims().size()));
        Tensor out_grad_3d_slice =
            out_grad_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                out_grad_3d.dims(), 1, out_grad_3d.dims().size()));
        Tensor col_buffer_3d_slice =
            col_buffer_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                col_buffer_3d.dims(), 1, col_buffer_3d.dims().size()));
        blas.MatMul(weight_3d_slice, true, out_grad_3d_slice, false, T(1.0),
                    &col_buffer_3d_slice, T(0.0));
      }
      modulated_deformable_col2im_coord(
          ctx.device_context(), col_buffer.data<T>(),
          input->data<T>() + i * im2col_step * input_dim,
          offset.data<T>() + i * im2col_step * input_offset_dim,
          mask.data<T>() + i * im2col_step * input_mask_dim, input_shape_vec,
          col_buffer_shape_vec, filter_shape_vec, paddings, strides, dilations,
          deformable_groups, offset_grad->mutable_data<T>(ctx.GetPlace()) +
                                 i * im2col_step * input_offset_dim,
          mask_grad->mutable_data<T>(ctx.GetPlace()) +
              i * im2col_step * input_mask_dim);

      modulated_deformable_col2im(
          ctx.device_context(), col_buffer.data<T>(),
          offset.data<T>() + i * im2col_step * input_offset_dim,
          mask.data<T>() + i * im2col_step * input_mask_dim, input_shape_vec,
          col_buffer_shape_vec, filter_shape_vec, paddings, strides, dilations,
          deformable_groups, col_buffer.mutable_data<T>(ctx.GetPlace()));

      modulated_deformable_im2col(
          ctx.device_context(), input->data<T>() + i * im2col_step * input_dim,
          offset.data<T>() + i * im2col_step * input_offset_dim,
          mask.data<T>() + i * im2col_step * input_mask_dim, input_shape_vec,
          col_buffer_shape_vec, filter_shape_vec, paddings, strides, dilations,
          deformable_groups, col_buffer.mutable_data<T>(ctx.GetPlace()));

      for (int g = 0; g < groups; g++) {
        Tensor out_grad_3d_slice =
            out_grad_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                out_grad_3d.dims(), 1, out_grad_3d.dims().size()));
        Tensor col_buffer_3d_slice =
            col_buffer_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                col_buffer_3d.dims(), 1, col_buffer_3d.dims().size()));
        Tensor dweight_3d_slice =
            dweight_3d.Slice(g, g + 1).Resize(framework::slice_ddim(
                dweight_3d.dims(), 1, dweight_3d.dims().size()));
        blas.MatMul(out_grad_3d_slice, false, col_buffer_3d_slice, true, T(1.0),
                    &dweight_3d_slice, T(0.0));
      }
    }
    // bias
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;

REGISTER_OP_CUDA_KERNEL(modulated_deformable_conv,
                        ops::ModulatedDeformableConvCUDAKernel<CUDA, float>);
// ops::ModulatedDeformableConvCUDAKernel<CUDA, double>);
REGISTER_OP_CUDA_KERNEL(
    modulated_deformable_conv_grad,
    ops::ModulatedDeformableConvGradCUDAKernel<CUDA, float>);
// ops::ModulatedDeformableConvGradCUDAKernel<CUDA, double>);
