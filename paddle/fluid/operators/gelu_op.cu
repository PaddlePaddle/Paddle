#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/gelu_op.h"

DECLARE_bool(use_fast_math);

namespace paddle {
namespace operators {

#ifdef __NVCC__
template <bool FastMode>
static __device__ __forceinline__ float FP32FastTanh(float x) {
#if __CUDA_ARCH__ >= 750 && CUDA_VERSION >= 11000
  if (FastMode) {
    float y;
    asm("tanh.approx.f32 %0,%1; \n\t" : "=f"(y) : "f"(x));
    return y;
  }
#endif
  return tanhf(x);
}

template <bool FastMode>
static __device__ __forceinline__ float FP32GeluFwd(float x) {
  auto tanh_out =
      FP32FastTanh<FastMode>(0.79788456f * x * (1.0f + 0.044715f * x * x));
  return x * 0.5f * (1.0f + tanh_out);
}

template <bool FastMode>
static __device__ __forceinline__ float FP32GeluBwd(float x, float y_g) {
  auto tanh_out =
      FP32FastTanh<FastMode>(0.79788456f * x * (1.0f + 0.044715f * x * x));
  auto tmp = 0.5f * x * ((1.0f - tanh_out * tanh_out) *
                         (0.79788456f + 0.1070322243f * x * x)) +
             0.5f * (1.0f + tanh_out);
  return tmp * y_g;
}

template <int VecSize, bool FastMode>
static __global__ void FP16FastGeluFwdCUDAKernel(const __half* x, __half* y,
                                                 size_t n) {
  size_t offset =
      static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x) * VecSize;
  size_t stride = static_cast<size_t>(blockDim.x * gridDim.x) * VecSize;
  for (; offset < n; offset += stride) {
    using ArrT = platform::AlignedVector<__half, VecSize>;
    ArrT in_arr = *reinterpret_cast<const ArrT*>(x + offset);
#pragma unroll
    for (int i = 0; i < VecSize; ++i) {
      float tmp = __half2float(in_arr[i]);
      in_arr[i] = __float2half(FP32GeluFwd<FastMode>(tmp));
    }
    *reinterpret_cast<ArrT*>(y + offset) = in_arr;
  }
}

template <int VecSize, bool FastMode>
static __global__ void FP16FastGeluBwdCUDAKernel(const __half* x,
                                                 const __half* y_g, __half* x_g,
                                                 size_t n) {
  size_t offset =
      static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x) * VecSize;
  size_t stride = static_cast<size_t>(blockDim.x * gridDim.x) * VecSize;
  for (; offset < n; offset += stride) {
    using ArrT = platform::AlignedVector<__half, VecSize>;
    ArrT x_in_arr = *reinterpret_cast<const ArrT*>(x + offset);
    ArrT y_g_in_arr = *reinterpret_cast<const ArrT*>(y_g + offset);
#pragma unroll
    for (int i = 0; i < VecSize; ++i) {
      __half2 tmp_fp16_2;
      tmp_fp16_2.x = x_in_arr[i];
      tmp_fp16_2.y = y_g_in_arr[i];
      float2 tmp_fp32_2 = __half22float2(tmp_fp16_2);
      x_in_arr[i] =
          __float2half(FP32GeluBwd<FastMode>(tmp_fp32_2.x, tmp_fp32_2.y));
    }
    *reinterpret_cast<ArrT*>(x_g + offset) = x_in_arr;
  }
}

static bool TryLaunchFP16FastGeluFwdVectorizeCUDAKernel(
    const platform::CUDADeviceContext& dev_ctx, const __half* x, __half* y,
    size_t n) {
  auto is_aligned = [](const void* p, size_t alignment) {
    return reinterpret_cast<uintptr_t>(p) % alignment == 0;
  };

#define PD_LAUNCH_FP16_FAST_GELU_FWD_KERNEL(__vec_size, __use_fast_math)      \
  do {                                                                        \
    constexpr auto kAlignment =                                               \
        alignof(platform::AlignedVector<__half, __vec_size>);                 \
    if (n % __vec_size == 0 && is_aligned(x, kAlignment) &&                   \
        is_aligned(y, kAlignment)) {                                          \
      size_t thread = std::min<size_t>(512, dev_ctx.GetMaxThreadsPerBlock()); \
      size_t block = (n / __vec_size + thread - 1) / thread;                  \
      block = std::min<size_t>(block, dev_ctx.GetCUDAMaxGridDimSize()[0]);    \
      VLOG(10) << "Use FP16 fast gelu fwd kernel, block = " << block          \
               << " , thread = " << thread;                                   \
      FP16FastGeluFwdCUDAKernel<                                              \
          __vec_size,                                                         \
          __use_fast_math><<<block, thread, 0, dev_ctx.stream()>>>(x, y, n);  \
      return true;                                                            \
    }                                                                         \
  } while (0)

  if (FLAGS_use_fast_math) {
    PD_LAUNCH_FP16_FAST_GELU_FWD_KERNEL(8, true);
  } else {
    PD_LAUNCH_FP16_FAST_GELU_FWD_KERNEL(8, false);
  }

#undef PD_LAUNCH_FP16_FAST_GELU_FWD_KERNEL
  return false;
}

static bool TryLaunchFP16FastGeluBwdVectorizeCUDAKernel(
    const platform::CUDADeviceContext& dev_ctx, const __half* x,
    const __half* y_g, __half* x_g, size_t n) {
  auto is_aligned = [](const void* p, size_t alignment) {
    return reinterpret_cast<uintptr_t>(p) % alignment == 0;
  };

#define PD_LAUNCH_FP16_FAST_GELU_BWD_KERNEL(__vec_size, __use_fast_math)      \
  do {                                                                        \
    constexpr auto kAlignment =                                               \
        alignof(platform::AlignedVector<__half, __vec_size>);                 \
    if (n % __vec_size == 0 && is_aligned(x, kAlignment) &&                   \
        is_aligned(x, kAlignment) && is_aligned(y_g, kAlignment) &&           \
        is_aligned(x_g, kAlignment)) {                                        \
      size_t thread = std::min<size_t>(512, dev_ctx.GetMaxThreadsPerBlock()); \
      size_t block = (n / __vec_size + thread - 1) / thread;                  \
      block = std::min<size_t>(block, dev_ctx.GetCUDAMaxGridDimSize()[0]);    \
      VLOG(10) << "Use FP16 fast gelu bwd kernel, block = " << block          \
               << " , thread = " << thread;                                   \
      FP16FastGeluBwdCUDAKernel<                                              \
          __vec_size,                                                         \
          __use_fast_math><<<block, thread, 0, dev_ctx.stream()>>>(x, y_g,    \
                                                                   x_g, n);   \
      return true;                                                            \
    }                                                                         \
  } while (0)

  if (FLAGS_use_fast_math) {
    PD_LAUNCH_FP16_FAST_GELU_BWD_KERNEL(8, true);
  } else {
    PD_LAUNCH_FP16_FAST_GELU_BWD_KERNEL(8, false);
  }

#undef PD_LAUNCH_FP16_FAST_GELU_BWD_KERNEL
  return false;
}
#endif

template <typename T>
struct GeluWithApproximateFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // this function is tanh approximation of gelu
    MPType x = static_cast<MPType>(arg_x);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    auto tanh_out =
        tanh(kAlpha * x * (one + static_cast<MPType>(GELU_CONSTANT) * x * x));
    MPType out = x * half * (one + tanh_out);
    return static_cast<T>(out);
  }
};

template <typename T>
struct GeluWithoutApproximateFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // actual gelu with approximation = false
    MPType x = static_cast<MPType>(arg_x);
    return static_cast<T>(x * normcdf(x));
  }
};

template <typename T>
class GeluKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* out = context.Output<framework::Tensor>("Out");
    auto* in = context.Input<framework::Tensor>("X");
    auto approximate = context.Attr<bool>("approximate");
    out->mutable_data<T>(in->place());

    std::vector<const framework::Tensor*> ins = {in};
    std::vector<framework::Tensor*> outs = {out};
    const auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();

    if (approximate) {
#ifdef __NVCC__
      if (std::is_same<T, platform::float16>::value) {
        size_t n = in->numel();
        const auto* in_ptr = reinterpret_cast<const __half*>(in->data<T>());
        auto* out_ptr = reinterpret_cast<__half*>(out->data<T>());
        if (TryLaunchFP16FastGeluFwdVectorizeCUDAKernel(dev_ctx, in_ptr,
                                                        out_ptr, n)) {
          return;
        }
      }
#endif
      paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary,
                                                     T, T>(
          dev_ctx, ins, &outs, 0, GeluWithApproximateFunctor<T>());
    } else {
      paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary,
                                                     T, T>(
          dev_ctx, ins, &outs, 0, GeluWithoutApproximateFunctor<T>());
    }
  }
};

template <typename T>
struct GeluWithApproximateGradFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x, T arg_dout) {
    MPType x = static_cast<MPType>(arg_x);
    MPType dout = static_cast<MPType>(arg_dout);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    MPType kBeta =
        kAlpha * static_cast<MPType>(GELU_CONSTANT) * static_cast<MPType>(3);
    auto cube_x = x * x * x;
    auto tanh_out =
        tanh(kAlpha * ((static_cast<MPType>(GELU_CONSTANT) * cube_x) + x));
    auto ans =
        half * (one + tanh_out +
                (one - tanh_out * tanh_out) * (x * kAlpha + kBeta * cube_x));
    return static_cast<T>(ans * dout);
  }
};

template <typename T>
struct GeluWithoutApproximateGradFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x, T arg_dout) {
    MPType x = static_cast<MPType>(arg_x);
    MPType dout = static_cast<MPType>(arg_dout);
    constexpr MPType kBeta = M_2_SQRTPI * M_SQRT1_2 * static_cast<MPType>(0.5);
    const MPType cdf = normcdf(x);
    const MPType pdf = exp(static_cast<MPType>(-0.5) * x * x) * kBeta;
    return static_cast<T>(dout * (cdf + x * pdf));
  }
};

template <typename T>
class GeluGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<framework::Tensor>("X");
    auto* dout =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* dx = context.Output<framework::Tensor>(framework::GradVarName("X"));
    auto approximate = context.Attr<bool>("approximate");
    dx->mutable_data<T>(dout->place());

    std::vector<const framework::Tensor*> ins = {x, dout};
    std::vector<framework::Tensor*> outs = {dx};
    const auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    if (approximate) {
#ifdef __NVCC__
      if (std::is_same<T, platform::float16>::value) {
        size_t n = x->numel();
        const auto* x_ptr = reinterpret_cast<const __half*>(x->data<T>());
        const auto* y_g_ptr = reinterpret_cast<const __half*>(dout->data<T>());
        auto* x_g_ptr = reinterpret_cast<__half*>(dx->data<T>());
        if (TryLaunchFP16FastGeluBwdVectorizeCUDAKernel(dev_ctx, x_ptr, y_g_ptr,
                                                        x_g_ptr, n)) {
          return;
        }
      }
#endif
      paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary,
                                                     T, T>(
          dev_ctx, ins, &outs, 0, GeluWithApproximateGradFunctor<T>());
    } else {
      paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary,
                                                     T, T>(
          dev_ctx, ins, &outs, 0, GeluWithoutApproximateGradFunctor<T>());
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    gelu, ops::GeluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GeluKernel<paddle::platform::CUDADeviceContext, double>,
    ops::GeluKernel<paddle::platform::CUDADeviceContext,
                    paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    gelu_grad, ops::GeluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GeluGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::GeluGradKernel<paddle::platform::CUDADeviceContext,
                        paddle::platform::float16>);
