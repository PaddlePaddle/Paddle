#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/gelu_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct GeluWithApproximateFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // this function is tanh approximation of gelu
    MPType x = static_cast<MPType>(arg_x);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType decimal = static_cast<MPType>(0.044715);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    auto tanh_out = tanh(kAlpha * x * (one + decimal * x * x));
    MPType out = x * half * (one + tanh_out);
    return static_cast<T>(out);
  }
};

template <typename T>
struct GeluWithoutApproximateFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // actual gelu with approximation = false
    MPType x = static_cast<MPType>(arg_x);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType erf_out = erf(x * static_cast<MPType>(M_SQRT1_2));
    MPType out = x * half * (one + erf_out);
    return static_cast<T>(out);
  }
};

template <typename T>
class GeluKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* out = context.Output<framework::Tensor>("Out");
    auto* in = context.Input<framework::Tensor>("X");
    auto approximate = context.Attr<bool>("approximate");
    out->mutable_data<T>(in->place());

    std::vector<const framework::Tensor*> ins = {in};
    std::vector<framework::Tensor*> outs = {out};
    const auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    if (approximate) {
      LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
          dev_ctx, ins, &outs, 0, GeluWithApproximateFunctor<T>());
    } else {
      LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
          dev_ctx, ins, &outs, 0, GeluWithoutApproximateFunctor<T>());
    }
  }
};

template <typename T>
struct GeluWithApproximateGradFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x, T arg_dout) {
    MPType x = static_cast<MPType>(arg_x);
    MPType dout = static_cast<MPType>(arg_dout);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType decimal = static_cast<MPType>(0.044715);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    MPType kBeta = kAlpha * decimal * static_cast<MPType>(3);
    auto tanh_out = tanh(kAlpha * x * (one + decimal * x * x));
    auto temp = (one - tanh_out * tanh_out) * (kAlpha + kBeta * x * x);
    auto ans = half * x * temp + half * (one + tanh_out);
    return static_cast<T>(ans * dout);
  }
};

template <typename T>
struct GeluWithoutApproximateGradFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x, T arg_dout) {
    MPType x = static_cast<MPType>(arg_x);
    MPType dout = static_cast<MPType>(arg_dout);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    auto ans = half * (one + erf(x * static_cast<MPType>(M_SQRT1_2))) +
               half * kAlpha * x * exp(-half * x * x);
    return static_cast<T>(ans * dout);
  }
};

template <typename T>
class GeluGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<framework::Tensor>("X");
    auto* dout =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* dx = context.Output<framework::Tensor>(framework::GradVarName("X"));
    auto approximate = context.Attr<bool>("approximate");
    dx->mutable_data<T>(dout->place());

    std::vector<const framework::Tensor*> ins = {x, dout};
    std::vector<framework::Tensor*> outs = {dx};
    const auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    if (approximate) {
      LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
          dev_ctx, ins, &outs, 0, GeluWithApproximateGradFunctor<T>());
    } else {
      LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
          dev_ctx, ins, &outs, 0, GeluWithoutApproximateGradFunctor<T>());
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    gelu, ops::GeluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GeluKernel<paddle::platform::CUDADeviceContext, double>,
    ops::GeluKernel<paddle::platform::CUDADeviceContext,
                    paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    gelu_grad, ops::GeluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GeluGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::GeluGradKernel<paddle::platform::CUDADeviceContext,
                        paddle::platform::float16>);
