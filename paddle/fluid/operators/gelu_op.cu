#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/gelu_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
struct GeluWithApproximateFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // this function is tanh approximation of gelu
    MPType x = static_cast<MPType>(arg_x);
    MPType one = static_cast<MPType>(1);
    MPType out = x * static_cast<MPType>(0.5) *
                 (one + tanh(static_cast<MPType>(0.79788456) * x *
                             (one + static_cast<MPType>(0.044715) * x * x)));
    return static_cast<T>(out);
  }
};

template <typename T>
struct GeluWithoutApproximateFunctor {
  using MPType = typename details::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // actual gelu with approximation = false
    MPType x = static_cast<MPType>(arg_x);
    MPType erf_out = erf(x * static_cast<MPType>(M_SQRT1_2));
    MPType out =
        x * static_cast<MPType>(0.5) * (static_cast<MPType>(1) + erf_out);
    return static_cast<T>(out);
  }
};

template <typename T>
class GeluKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* out = context.Output<framework::Tensor>("Out");
    auto* in = context.Input<framework::Tensor>("X");
    auto approximate = context.Attr<bool>("approximate");
    out->mutable_data<T>(in->place());

    std::vector<const framework::Tensor*> ins = {in};
    std::vector<framework::Tensor*> outs = {out};
    const auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    if (approximate) {
      LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
          dev_ctx, ins, &outs, 0, GeluWithApproximateFunctor<T>());
    } else {
      LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
          dev_ctx, ins, &outs, 0, GeluWithoutApproximateFunctor<T>());
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    gelu, ops::GeluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GeluKernel<paddle::platform::CUDADeviceContext, double>,
    ops::GeluKernel<paddle::platform::CUDADeviceContext,
                    paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    gelu_grad, ops::GeluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GeluGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::GeluGradKernel<paddle::platform::CUDADeviceContext,
                        paddle::platform::float16>);
