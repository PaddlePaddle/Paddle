#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
Indicesou may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#define EIGEN_USE_GPU

#include "paddle/fluid/operators/norm_op.h"

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    norm, ops::NormKernel<paddle::platform::CUDADeviceContext, float>,
    ops::NormKernel<paddle::platform::CUDADeviceContext, double, float>);
REGISTER_OP_CUDA_KERNEL(
    norm_grad, ops::NormGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::NormGradKernel<paddle::platform::CUDADeviceContext, double, float>);
