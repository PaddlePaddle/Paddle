#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
Indicesou may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/operators/norm_op.h"

namespace paddle {
namespace operators {

__device__ __forceinline__ float square_root(float x) { return sqrtf(x); }

__device__ __forceinline__ double square_root(double x) { return sqrt(x); }

template <typename T, int BlockDim>
__global__ void Normalize(const T* x, const int pre,
                          const int axis_n,  // dim in axis
                          const int post, const T eps, T* y, T* out_norm) {
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int num = pre * post;
  for (int i = blockIdx.x; i < num; i += gridDim.x) {
    int base = (i / post) * post * axis_n + (i % post);

    T sum = 0.0;
    __shared__ T norm;
    for (int j = threadIdx.x; j < axis_n; j += blockDim.x) {
      const T x_ij = x[base + j * post];
      sum += x_ij * x_ij;
    }
    T reduce_result = BlockReduce(temp_storage).Sum(sum);

    if (threadIdx.x == 0) {
      norm = square_root(reduce_result + eps);
      out_norm[i] = norm;
    }
    __syncthreads();
    for (int j = threadIdx.x; j < axis_n; j += blockDim.x) {
      const int index = base + j * post;
      y[index] = x[index] / norm;
    }
  }
}

template <typename DeviceContext, typename T>
class NormCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in_x = ctx.Input<framework::Tensor>("X");
    auto* out_y = ctx.Output<framework::Tensor>("Out");
    auto* out_norm = ctx.Output<framework::Tensor>("Norm");
    const T* x = in_x->data<T>();
    T* y = out_y->mutable_data<T>(ctx.GetPlace());
    T* norm = out_norm->mutable_data<T>(ctx.GetPlace());

    auto xdim = in_x->dims();
    auto ndim = out_norm->dims();
    int axis = ctx.Attr<int>("axis");
    T eps = static_cast<T>(ctx.Attr<float>("epsilon"));
    if (axis < 0) axis = xdim.size() + axis;
    int pre, n, post;
    GetDims(xdim, axis, &pre, &n, &post);

    auto& dev_ctx = ctx.cuda_device_context();

    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    int grid = std::min(max_blocks, pre * post);
    Normalize<T, block><<<grid, block, 0, dev_ctx.stream()>>>(x, pre, n, post,
                                                              eps, y, norm);
  }
};

template <typename T, int BlockDim>
__global__ void NormalizeGradient(const T* x, const T* x_norm, const T* y_grad,
                                  const int pre, const int axis_n,
                                  const int post, T* x_grad) {
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage_sum;
  int num = pre * post;
  for (int i = blockIdx.x; i < num; i += gridDim.x) {
    T sum = 0.0;
    __shared__ T row_sum;
    __shared__ T row_sqrt_norm;
    __shared__ T row_norm;

    auto base = (i / post) * post * axis_n + (i % post);

    for (int j = threadIdx.x; j < axis_n; j += blockDim.x) {
      int index = base + j * post;
      sum += x[index] * y_grad[index];
    }
    T reduce_result = BlockReduce(temp_storage_sum).Sum(sum);

    if (threadIdx.x == 0) {
      row_sum = reduce_result;
      row_sqrt_norm = x_norm[i];
      row_norm = row_sqrt_norm * row_sqrt_norm;
    }
    __syncthreads();
    for (int j = threadIdx.x; j < axis_n; j += blockDim.x) {
      int index = base + j * post;
      const T x_ij = x[index];
      const T dy_ij = y_grad[index];
      x_grad[index] = (dy_ij - x_ij * row_sum / row_norm) / row_sqrt_norm;
    }
  }
}

template <typename DeviceContext, typename T, typename AttrType = T>
class NormGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in_x = ctx.Input<framework::Tensor>("X");
    auto* in_norm = ctx.Input<framework::Tensor>("Norm");
    auto* in_dy = ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* out_dx = ctx.Output<framework::Tensor>(framework::GradVarName("X"));
    T* dx = out_dx->mutable_data<T>(ctx.GetPlace());
    const T* x = in_x->data<T>();
    const T* x_norm = in_norm->data<T>();
    const T* dy = in_dy->data<T>();

    auto xdim = in_x->dims();
    int axis = ctx.Attr<int>("axis");
    if (axis < 0) axis = xdim.size() + axis;
    int pre, n, post;
    GetDims(xdim, axis, &pre, &n, &post);

    auto& dev_ctx = ctx.cuda_device_context();

    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    int grid = std::min(max_blocks, pre * post);
    NormalizeGradient<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
        x, x_norm, dy, pre, n, post, dx);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;

REGISTER_OP_CUDA_KERNEL(norm, ops::NormCUDAKernel<CUDA, float>,
                        ops::NormCUDAKernel<CUDA, double>);
REGISTER_OP_CUDA_KERNEL(norm_grad, ops::NormGradCUDAKernel<CUDA, float>,
                        ops::NormGradCUDAKernel<CUDA, double>);
