#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <random>
#include <vector>

#include "gtest/gtest.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/framework/program_desc.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/fused/attn_feed_forward.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace framework = paddle::framework;
namespace platform = paddle::platform;

USE_OP(matmul);
USE_OP_ITSELF(elementwise_add);

// get paddle matmul op results as baseline
template <typename T>
void GetLinearOp(const std::vector<T> &x, const std::vector<T> &y,
                 const framework::DDim &x_dim, const framework::DDim &y_dim,
                 const platform::CUDADeviceContext &ctx, bool transpose_a,
                 bool transpose_b, float alpha, std::vector<T> *out) {
  framework::Scope scope;
  auto var_x = scope.Var("X");
  auto tensor_x = var_x->GetMutable<framework::LoDTensor>();
  auto var_y = scope.Var("Y");
  auto tensor_y = var_y->GetMutable<framework::LoDTensor>();
  auto var_out = scope.Var("Out");
  auto tensor_out = var_out->GetMutable<framework::LoDTensor>();

  tensor_x->Resize(x_dim);
  tensor_y->Resize(y_dim);
  tensor_out->Resize({x_dim[0], x_dim[1], y_dim[0]});

  auto x_ptr = tensor_x->mutable_data<T>(ctx.GetPlace());
  auto y_ptr = tensor_y->mutable_data<T>(ctx.GetPlace());
  auto z_ptr = tensor_out->mutable_data<T>(ctx.GetPlace());
  auto size_x = static_cast<size_t>(phi::product(x_dim));
  auto size_y = static_cast<size_t>(phi::product(y_dim));
  auto size_z = x_dim[0] * x_dim[1] * y_dim[0];
  hipMemcpy(x_ptr, x.data(), size_x * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(y_ptr, y.data(), size_y * sizeof(T), hipMemcpyHostToDevice);

  framework::AttributeMap attrs;
  attrs.insert({"transpose_X", transpose_a});
  attrs.insert({"transpose_Y", transpose_b});
  attrs.insert({"alpha", alpha});

  auto op = framework::OpRegistry::CreateOp(
      "matmul", {{"X", {"X"}}, {"Y", {"Y"}}}, {{"Out", {"Out"}}}, attrs);
  op->Run(scope, ctx.GetPlace());

  hipMemcpy(out->data(), z_ptr, size_z * sizeof(T), hipMemcpyDeviceToHost);
  ctx.Wait();
}

// get paddle elementwise_add op results as baseline
template <typename T>
void GetElementwiseAddOp(const std::vector<T> &x, const std::vector<T> &y,
                         const int bsz_seq, const int output_size,
                         const platform::CUDADeviceContext &ctx,
                         std::vector<T> *out) {
  framework::Scope scope;
  auto var_x = scope.Var("X");
  auto tensor_x = var_x->GetMutable<framework::LoDTensor>();
  auto var_y = scope.Var("Y");
  auto tensor_y = var_y->GetMutable<framework::LoDTensor>();
  auto var_out = scope.Var("Out");
  auto tensor_out = var_out->GetMutable<framework::LoDTensor>();

  tensor_x->Resize({bsz_seq, output_size});
  tensor_y->Resize({output_size});
  tensor_out->Resize({bsz_seq, output_size});

  auto x_ptr = tensor_x->mutable_data<T>(ctx.GetPlace());
  auto y_ptr = tensor_y->mutable_data<T>(ctx.GetPlace());
  auto z_ptr = tensor_out->mutable_data<T>(ctx.GetPlace());
  auto size_x = bsz_seq * output_size;
  auto size_y = output_size;
  auto size_z = bsz_seq * output_size;
  hipMemcpy(x_ptr, x.data(), size_x * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(y_ptr, y.data(), size_y * sizeof(T), hipMemcpyHostToDevice);

  framework::AttributeMap attrs;
  auto op = framework::OpRegistry::CreateOp("elementwise_add",
                                            {{"X", {"X"}}, {"Y", {"Y"}}},
                                            {{"Out", {"Out"}}}, attrs);
  op->Run(scope, ctx.GetPlace());
  hipMemcpy(out->data(), z_ptr, size_z * sizeof(T), hipMemcpyDeviceToHost);
  ctx.Wait();
}

// get paddle matmul_grad op results as baseline
template <typename T>
void GetLinearOpGrad(const std::vector<T> &x_vec, const std::vector<T> &y_vec,
                     const std::vector<T> &dout_vec,
                     const framework::DDim &x_dim, const framework::DDim &y_dim,
                     const framework::DDim &out_dim,
                     const platform::CUDADeviceContext &ctx, bool transpose_a,
                     bool transpose_b, float alpha, std::vector<T> *dinput_vec,
                     std::vector<T> *dweight_vec) {
  framework::Scope scope;
  auto var_x = scope.Var("X");
  auto tensor_x = var_x->GetMutable<framework::LoDTensor>();
  auto var_y = scope.Var("Y");
  auto tensor_y = var_y->GetMutable<framework::LoDTensor>();
  auto var_dout = scope.Var("DOut");
  auto tensor_dout = var_dout->GetMutable<framework::LoDTensor>();
  tensor_x->Resize(x_dim);
  tensor_y->Resize(y_dim);
  tensor_dout->Resize(out_dim);

  auto var_dx = scope.Var("DX");
  auto tensor_dx = var_dx->GetMutable<framework::LoDTensor>();
  auto var_dy = scope.Var("DY");
  auto tensor_dy = var_dy->GetMutable<framework::LoDTensor>();
  tensor_dx->Resize(x_dim);
  tensor_dy->Resize(y_dim);

  auto x_ptr = tensor_x->mutable_data<T>(ctx.GetPlace());
  auto y_ptr = tensor_y->mutable_data<T>(ctx.GetPlace());
  auto dout_ptr = tensor_dout->mutable_data<T>(ctx.GetPlace());
  auto dinput_ptr = tensor_dx->mutable_data<T>(ctx.GetPlace());
  auto dweight_ptr = tensor_dy->mutable_data<T>(ctx.GetPlace());

  auto size_x = static_cast<size_t>(phi::product(x_dim));
  auto size_y = static_cast<size_t>(phi::product(y_dim));
  auto size_z = x_dim[0] * x_dim[1] * y_dim[0];
  hipMemcpy(x_ptr, x_vec.data(), size_x * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(y_ptr, y_vec.data(), size_y * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(dout_ptr, dout_vec.data(), size_z * sizeof(T),
             hipMemcpyHostToDevice);

  bool use_mkldnn = false;
  std::vector<int> fused_reshape_X = {};
  std::vector<int> fused_reshape_Y = {};
  std::vector<int> fused_reshape_Out = {};
  std::vector<int> fused_transpose_X = {};
  std::vector<int> fused_transpose_Y = {};
  std::vector<int> fused_transpose_Out = {};
  bool use_quantizer = false, force_fp32_output = false;
  std::string mkldnn_data_type = "float32";
  float Scale_x = 1.0, Scale_y = 1.0, Scale_out = 1.0;

  framework::AttributeMap attrs;
  attrs.insert({"transpose_X", transpose_a});
  attrs.insert({"transpose_Y", transpose_b});
  attrs.insert({"alpha", alpha});
  attrs.insert({"use_mkldnn", use_mkldnn});
  attrs.insert({"fused_reshape_X", fused_reshape_X});
  attrs.insert({"fused_reshape_Y", fused_reshape_Y});
  attrs.insert({"fused_reshape_Out", fused_reshape_Out});
  attrs.insert({"fused_transpose_X", fused_transpose_X});
  attrs.insert({"fused_transpose_Y", fused_transpose_Y});
  attrs.insert({"fused_transpose_Out", fused_transpose_Out});
  attrs.insert({"use_quantizer", use_quantizer});
  attrs.insert({"mkldnn_data_type", mkldnn_data_type});
  attrs.insert({"Scale_x", Scale_x});
  attrs.insert({"Scale_y", Scale_y});
  attrs.insert({"Scale_out", Scale_out});
  attrs.insert({"force_fp32_output", force_fp32_output});

  auto op = framework::OpRegistry::CreateOp(
      "matmul_grad", {{"Out@GRAD", {"DOut"}}, {"X", {"X"}}, {"Y", {"Y"}}},
      {{"X@GRAD", {"DX"}}, {"Y@GRAD", {"DY"}}}, attrs);
  op->Run(scope, ctx.GetPlace());

  hipMemcpy(dinput_vec->data(), dinput_ptr, size_x * sizeof(T),
             hipMemcpyDeviceToHost);
  hipMemcpy(dweight_vec->data(), dweight_ptr, size_y * sizeof(T),
             hipMemcpyDeviceToHost);
  ctx.Wait();
}

// get paddle elementwise_add_grad op results as baseline
template <typename T>
void GetElementwiseAddOpGrad(const std::vector<T> &dout_vec, const int bsz_seq,
                             const int output_size,
                             const platform::CUDADeviceContext &ctx,
                             std::vector<T> *dy_vec) {
  framework::Scope scope;
  auto var_x = scope.Var("X");
  auto tensor_x = var_x->GetMutable<framework::LoDTensor>();
  auto var_y = scope.Var("Y");
  auto tensor_y = var_y->GetMutable<framework::LoDTensor>();
  auto var_dout = scope.Var("DOut");
  auto tensor_dout = var_dout->GetMutable<framework::LoDTensor>();
  tensor_x->Resize({bsz_seq, output_size});
  tensor_y->Resize({output_size});
  tensor_dout->Resize({bsz_seq, output_size});

  auto var_dx = scope.Var("DX");
  auto tensor_dx = var_dx->GetMutable<framework::LoDTensor>();
  auto var_dy = scope.Var("DY");
  auto tensor_dy = var_dy->GetMutable<framework::LoDTensor>();
  tensor_dx->Resize({bsz_seq, output_size});
  tensor_dy->Resize({output_size});

  auto dout_ptr = tensor_dout->mutable_data<T>(ctx.GetPlace());
  auto tensor_dy_ptr = tensor_dy->mutable_data<T>(ctx.GetPlace());
  auto size_z = static_cast<size_t>(bsz_seq * output_size);
  hipMemcpy(dout_ptr, dout_vec.data(), size_z * sizeof(T),
             hipMemcpyHostToDevice);

  int axis = -1;
  bool use_mkldnn = false, use_quantizer = false;
  std::string mkldnn_data_type = "float32";
  std::string x_data_format = "", y_data_format = "";
  float Scale_x = 1.0, Scale_y = 1.0, Scale_out = 1.0;

  framework::AttributeMap attrs;
  attrs.insert({"axis", axis});
  attrs.insert({"use_mkldnn", use_mkldnn});
  attrs.insert({"x_data_format", x_data_format});
  attrs.insert({"y_data_format", y_data_format});
  attrs.insert({"use_quantizer", use_quantizer});
  attrs.insert({"mkldnn_data_type", mkldnn_data_type});
  attrs.insert({"Scale_x", Scale_x});
  attrs.insert({"Scale_y", Scale_y});
  attrs.insert({"Scale_out", Scale_out});

  auto op = framework::OpRegistry::CreateOp(
      "elementwise_add_grad",
      {{"Out@GRAD", {"DOut"}}, {"X", {"X"}}, {"Y", {"Y"}}},
      {{"X@GRAD", {"DX"}}, {"Y@GRAD", {"DY"}}}, attrs);
  op->Run(scope, ctx.GetPlace());

  auto size_y = static_cast<size_t>(output_size);
  hipMemcpy(dy_vec->data(), tensor_dy_ptr, size_y * sizeof(T),
             hipMemcpyDeviceToHost);
  ctx.Wait();
}

template <typename T>
class TestFeedForward {
 public:
  TestFeedForward() {
    batch_size_ = 16;
    seq_len_ = 128;
    num_head_ = 16;
    dim_head_ = 64;
    dim_embed_ = 1024;
    has_bias_ = false;
  }

  TestFeedForward(int batch_size, int seq_len, int num_head, int dim_head,
                  int dim_embed, bool has_bias) {
    batch_size_ = batch_size;
    seq_len_ = seq_len;
    num_head_ = num_head;
    dim_head_ = dim_head;
    dim_embed_ = dim_embed;
    has_bias_ = has_bias;
  }

  ~TestFeedForward() { delete ctx_; }

  void SetUp() {
    bsz_seq_ = batch_size_ * seq_len_;
    output_size_ = 3 * num_head_ * dim_head_;
    input_size_ = dim_embed_;
    ctx_ = new platform::CUDADeviceContext(place_);
    ctx_->SetAllocator(paddle::memory::allocation::AllocatorFacade::Instance()
                           .GetAllocator(place_, ctx_->stream())
                           .get());
    ctx_->SetHostAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(paddle::platform::CPUPlace())
            .get());
    ctx_->SetZeroAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetZeroAllocator(place_)
            .get());
    ctx_->PartialInitWithAllocator();

    size_src_ = bsz_seq_ * dim_embed_;         // src: [bs, seq_len, em_dim]
    size_weight_ = dim_embed_ * output_size_;  // weight: [output_size, em_dim]
    size_output_ =
        bsz_seq_ * output_size_;  // output: [bs, seq_len, output_size]
    size_bias_ = output_size_;

    base_out_vec_.resize(size_output_);
    base_bias_out_vec_.resize(size_output_);
    base_dinput_vec_.resize(size_src_);
    base_dweight_vec_.resize(size_weight_);
    base_dbias_vec_.resize(size_bias_);

    src_vec_.resize(size_src_);
    weight_vec_.resize(size_weight_);
    bias_vec_.resize(size_bias_);
    doutput_vec_.resize(size_output_);

    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> dis(0.0, 1.0);
    for (int i = 0; i < size_src_; i++) {
      src_vec_[i] = static_cast<T>(dis(random));
    }
    for (int i = 0; i < size_weight_; i++) {
      weight_vec_[i] = static_cast<T>(dis(random));
    }
    for (int i = 0; i < size_bias_; i++) {
      bias_vec_[i] = static_cast<T>(dis(random));
    }
    for (int i = 0; i < size_output_; i++) {
      doutput_vec_[i] = static_cast<T>(dis(random));
    }

    framework::TensorFromVector<T>(src_vec_, *ctx_, &src_);
    src_.Resize({batch_size_, seq_len_, dim_embed_});
    framework::TensorFromVector<T>(weight_vec_, *ctx_, &weight_);
    weight_.Resize({output_size_, dim_embed_});
    out_.Resize({batch_size_, seq_len_, output_size_});
    out_.mutable_data<T>(place_);
    if (has_bias_) {
      framework::TensorFromVector<T>(bias_vec_, *ctx_, &bias_);
      bias_.Resize({output_size_});
      bias_out_.Resize({batch_size_, seq_len_, output_size_});
      bias_out_.mutable_data<T>(place_);
    }
    framework::TensorFromVector<T>(doutput_vec_, *ctx_, &doutput_);
    doutput_.Resize({batch_size_, seq_len_, output_size_});

    dinput_.Resize({batch_size_, seq_len_, dim_embed_});
    dinput_.mutable_data<T>(place_);
    dweight_.Resize({output_size_, dim_embed_});
    dweight_.mutable_data<T>(place_);
    if (has_bias_) {
      dbias_.Resize({output_size_});
      dbias_.mutable_data<T>(place_);
    }
  }

  void BaselineForward() {
    bool transpose_a = false, transpose_b = true;
    float alpha = 1;
    GetLinearOp(src_vec_, weight_vec_, src_.dims(), weight_.dims(), *ctx_,
                transpose_a, transpose_b, alpha, &base_out_vec_);
    if (has_bias_) {
      GetElementwiseAddOp(base_out_vec_, bias_vec_, bsz_seq_, output_size_,
                          *ctx_, &base_bias_out_vec_);
    }
    ctx_->Wait();
  }

  // get forward results of feedforward.
  void FusedForward() {
    T *p_weight = weight_.data<T>();
    T *p_src = src_.data<T>();
    T *p_output = out_.data<T>();

    T *p_bias = nullptr;
    T *p_bias_output = nullptr;
    if (has_bias_) {
      p_bias = bias_.data<T>();
      p_bias_output = bias_out_.data<T>();
    }
    auto qkv_compute = paddle::operators::FeedForward<T>(
        *ctx_, bsz_seq_, output_size_, input_size_, has_bias_);
    qkv_compute.ComputeForward(p_weight, p_src, p_bias, p_output,
                               p_bias_output);
    ctx_->Wait();
  }

  void BaselineBackward() {
    bool transpose_a = false, transpose_b = true;
    float alpha = 1;

    GetLinearOpGrad(src_vec_, weight_vec_, doutput_vec_, src_.dims(),
                    weight_.dims(), out_.dims(), *ctx_, transpose_a,
                    transpose_b, alpha, &base_dinput_vec_, &base_dweight_vec_);
    if (has_bias_) {
      GetElementwiseAddOpGrad(doutput_vec_, bsz_seq_, output_size_, *ctx_,
                              &base_dbias_vec_);
    }
    ctx_->Wait();
  }

  // get backward results of feedforward.
  void FusedBackward() {
    T *p_weight = weight_.data<T>();
    T *p_src = src_.data<T>();
    T *p_doutput = doutput_.data<T>();
    T *p_dinput = dinput_.data<T>();
    T *p_dweight = dweight_.data<T>();

    T *bias_ptr = nullptr;
    if (has_bias_) {
      bias_ptr = dbias_.data<T>();
    }
    auto qkv_compute = paddle::operators::FeedForward<T>(
        *ctx_, bsz_seq_, output_size_, input_size_, has_bias_);
    qkv_compute.ComputeBackward(p_src, p_weight, p_doutput, p_dinput, p_dweight,
                                bias_ptr);
    ctx_->Wait();
  }

  void Run() {
    SetUp();
    BaselineForward();
    FusedForward();
    BaselineBackward();
    FusedBackward();
  }

  // check forward correctness between baseline and results of feedforward.
  void CheckOut(const T diff, bool is_relative_atol = false) {
    std::vector<T> out(size_output_);
    std::vector<T> bias_out(size_output_);
    paddle::framework::TensorToVector(out_, *ctx_, &out);
    if (has_bias_) {
      paddle::framework::TensorToVector(bias_out_, *ctx_, &bias_out);
    }
    ctx_->Wait();

    for (int i = 0; i < size_output_; i++) {
      if (is_relative_atol) {
        EXPECT_LT(std::abs((out[i] - base_out_vec_[i]) / base_out_vec_[i]),
                  diff);
      } else {
        EXPECT_LT(std::abs(out[i] - base_out_vec_[i]), diff);
      }
      if (has_bias_) {
        if (is_relative_atol) {
          EXPECT_LT(std::abs((bias_out[i] - base_bias_out_vec_[i]) /
                             base_bias_out_vec_[i]),
                    diff);
        } else {
          EXPECT_LT(std::abs(bias_out[i] - base_bias_out_vec_[i]), diff);
        }
      }
    }
  }

  // check backward correctness between baseline and results of feedforward.
  void CheckGrad(const T diff, bool is_relative_atol = false) {
    std::vector<T> h_dinput(size_src_);
    paddle::framework::TensorToVector(dinput_, *ctx_, &h_dinput);
    for (int i = 0; i < size_src_; i++) {
      if (is_relative_atol) {
        EXPECT_LT(
            std::abs((h_dinput[i] - base_dinput_vec_[i]) / base_dinput_vec_[i]),
            diff);
      } else {
        EXPECT_LT(std::abs(h_dinput[i] - base_dinput_vec_[i]), diff);
      }
    }
    std::vector<T> h_dweight(size_weight_);
    paddle::framework::TensorToVector(dweight_, *ctx_, &h_dweight);
    for (int i = 0; i < size_weight_; i++) {
      if (is_relative_atol) {
        EXPECT_LT(std::abs((h_dweight[i] - base_dweight_vec_[i]) /
                           base_dweight_vec_[i]),
                  diff);
      } else {
        EXPECT_LT(std::abs(h_dweight[i] - base_dweight_vec_[i]), diff);
      }
    }
    if (has_bias_) {
      std::vector<T> h_dbias(size_bias_);
      paddle::framework::TensorToVector(dbias_, *ctx_, &h_dbias);
      for (int i = 0; i < size_bias_; i++) {
        if (is_relative_atol) {
          EXPECT_LT(
              std::abs((h_dbias[i] - base_dbias_vec_[i]) / base_dbias_vec_[i]),
              diff);
        } else {
          EXPECT_LT(std::abs(h_dbias[i] - base_dbias_vec_[i]), diff);
        }
      }
    }
  }

 private:
  int batch_size_, seq_len_, num_head_, dim_head_, dim_embed_;
  int bsz_seq_, output_size_, input_size_;
  bool has_bias_;
  int size_src_, size_weight_, size_bias_, size_output_;

  framework::Tensor src_, weight_, bias_, out_, bias_out_;
  framework::Tensor dinput_, dweight_, dbias_, doutput_;
  std::vector<T> src_vec_, weight_vec_, bias_vec_, out_vec_, bias_out_vec_;
  std::vector<T> dinput_vec_, dweight_vec_, dbias_vec_, doutput_vec_;

  // results of baseline.
  std::vector<T> base_out_vec_, base_bias_out_vec_;
  std::vector<T> base_dinput_vec_, base_dweight_vec_, base_dbias_vec_;

  platform::CUDAPlace place_;
  platform::CUDADeviceContext *ctx_;
};

// test for fp32, fp16, fp32+bias and fp16+bias
TEST(FeedForward, GPUFeedforwardBertLargeSizeFp32) {
  int batch_size = 16;
  int seq_len = 128;
  int num_head = 16;
  int dim_head = 64;
  int dim_embed = 1024;
  bool has_bias = false;
  TestFeedForward<float> test(batch_size, seq_len, num_head, dim_head,
                              dim_embed, has_bias);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FeedForward, GPUFeedforwardBertLargeSizeFp16) {
  int batch_size = 16;
  int seq_len = 128;
  int num_head = 16;
  int dim_head = 64;
  int dim_embed = 1024;
  bool has_bias = false;
  TestFeedForward<paddle::platform::float16> test(
      batch_size, seq_len, num_head, dim_head, dim_embed, has_bias);
  test.Run();
  test.CheckOut(static_cast<paddle::platform::float16>(1e-5));
  test.CheckGrad(static_cast<paddle::platform::float16>(1e-5));
}

TEST(FeedForward, GPUFeedforwardBertLargeSizeFp32Bias) {
  int batch_size = 16;
  int seq_len = 128;
  int num_head = 16;
  int dim_head = 64;
  int dim_embed = 1024;
  bool has_bias = true;
  TestFeedForward<float> test(batch_size, seq_len, num_head, dim_head,
                              dim_embed, has_bias);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-3));
}

TEST(FeedForward, GPUFeedforwardBertLargeSizeFp16Bias) {
  int batch_size = 16;
  int seq_len = 128;
  int num_head = 16;
  int dim_head = 64;
  int dim_embed = 1024;
  bool has_bias = true;
  TestFeedForward<paddle::platform::float16> test(
      batch_size, seq_len, num_head, dim_head, dim_embed, has_bias);
  test.Run();
  test.CheckOut(static_cast<paddle::platform::float16>(1e-2));
  test.CheckGrad(static_cast<paddle::platform::float16>(1e-2), true);
}
