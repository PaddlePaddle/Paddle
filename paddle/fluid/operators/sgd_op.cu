#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/fluid/operators/sgd_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

namespace {

template <typename T>
struct SGDFunctor {
  const T* g;
  const T* p;
  const T* learning_rate;
  T* p_out;
  SGDFunctor(const T* g, const T* p, const T* l, const int num, T* p_out)
      : g(g), p(p), learning_rate(l), num(num), p_out(p_out) {}

  inline HOSTDEVICE void operator()(size_t i) const {
    p_out[i] = p[i] - lr * g[i];
  }
};

template <typename T>
struct SGDWithReplicaKernel {
  const T* g;
  const float* p_replica;
  const T* learning_rate;
  T* p_out;
  float* p_replica_out;
  SGDWithReplicaKernel(const T* g, const float* p_replica, const T* l,
                       const int num, T* p_out, float* p_replica_out)
      : g(g),
        p_replica(p_replica),
        learning_rate(l),
        num(num),
        p_out(p_out),
        p_replica_out(p_replica_out) {}

  inline HOSTDEVICE void operator()(size_t i) const {
    p_replica_out[i] = p_replica[i] - lr.float() * g_data.float();
    p_out[i] = platform::float16(p_replica_out[i]);
  }
};

template <typename T, int block_size>
__global__ void SparseSGDFunctorKernel(const T* selected_rows,
                                       const int64_t* rows,
                                       const T* learning_rate, T* tensor_out,
                                       int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(
        tensor_out + index, -1.0 * learning_rate[0] * selected_rows[index]);
  }
}
}  // namespace

template <typename T>
class SGDOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* param = ctx.Input<framework::Tensor>("Param");
    auto* param_out = ctx.Output<framework::Tensor>("ParamOut");
    auto* learning_rate = ctx.Input<framework::Tensor>("LearningRate");

    auto* grad_var = ctx.InputVar("Grad");
    // Actually, all tensors are LoDTensor except SelectedRows.
    if (grad_var->IsType<framework::LoDTensor>()) {
      param_out->mutable_data<T>(ctx.GetPlace());
      auto* grad = ctx.Input<framework::Tensor>("Grad");

      // auto for_range(ctx.template device_context(), param->numel());
      platform::ForRange<platform::CUDADeviceContext> for_range(
          ctx.device_context(), param.numel());

      if (ctx.Attr<bool>("mixed_precision_mode")) {
        PADDLE_ENFORCE(std::type_index(typeid(T)) ==
                           std::type_index(typeid(platform::float16)),
                       "mixed_precision_mode is only supported in float16.");
        auto* param_replica = ctx.Input<framework::Tensor>("ParamReplica");
        auto* param_replica_out =
            ctx.Output<framework::Tensor>("ParamReplicaOut");
        SGDWithReplicaKernel<T> functor(
            grad->data<T>(), param_replica->data<T>(), learning_rate->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()),
            param_replica_out->mutable_data<T>(ctx.GetPlace()));
        for_range(functor);
      } else {
        SGDFunctor<T> functor(grad->data<T>(), param->data<T>(),
                              learning_rate->data<T>(),
                              param_out->mutable_data<T>(ctx.GetPlace()));
        for_range(functor);
      }

    } else if (grad_var->IsType<framework::SelectedRows>()) {
      // TODO(qijun): In Sparse SGD operator, in-place update is enforced.
      // This manual optimization brings difficulty to track data dependency.
      // It's better to find a more elegant solution.
      PADDLE_ENFORCE_EQ(param, param_out);
      auto* grad = ctx.Input<framework::SelectedRows>("Grad");

      auto in_height = grad->height();
      auto out_dims = param_out->dims();
      PADDLE_ENFORCE_EQ(in_height, out_dims[0]);

      auto& in_value = grad->value();
      framework::Vector<int64_t> in_rows(grad->rows());

      int64_t in_row_numel = in_value.numel() / in_rows.size();
      PADDLE_ENFORCE_EQ(in_row_numel, param_out->numel() / in_height);

      auto* in_data = in_value.data<T>();
      auto* out_data = param_out->data<T>();

      const int block_size = 256;
      dim3 threads(block_size, 1);
      dim3 grid(1, in_rows.size());
      SparseSGDFunctorKernel<
          T, 256><<<grid, threads, 0, ctx.cuda_device_context().stream()>>>(
          in_data, in_rows.CUDAData(ctx.GetPlace()), learning_rate->data<T>(),
          out_data, in_row_numel);

    } else {
      PADDLE_THROW("Unsupported Variable Type of Grad");
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(sgd, ops::SGDOpCUDAKernel<float>,
                        ops::SGDOpCUDAKernel<double>,
                        ops::SGDOpCUDAKernel<plat::float16>);
