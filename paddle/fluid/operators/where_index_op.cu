#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include <algorithm>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/where_index_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/kernels/primitive/kernel_primitives.h"

namespace paddle {
namespace operators {

using CUDADeviceContext = paddle::platform::CUDADeviceContext;
namespace kps = phi::kps;
using Mode = kps::details::ReduceMode;

/*
* Count how many of the data being processed by the current block are true
* 1. Load data from global memory and cast from bool to int64_t
* 2. Get result of this thread according to thread reduce
* 3. Get result of this block according to block reduce
* 4. first block store 0 and current result
*/

template <typename InT, typename OutT, int VecSize, int IsBoundary>
__device__ void GetBlockCountImpl(const InT *in, OutT *out, int num,
                                  int repeat) {
  InT in_data[VecSize];
  OutT temp[VecSize];
  OutT result = static_cast<OutT>(0.0f);
  using Add = kps::AddFunctor<OutT>;
  using Cast = kps::IdentityFunctor<InT>;
  int store_fix = BLOCK_ID_X + repeat * GRID_NUM_X;

  kps::Init<InT, VecSize>(&in_data[0], static_cast<InT>(0.0f));
  kps::ReadData<InT, VecSize, 1, 1, IsBoundary>(&in_data[0], in, num);
  kps::ElementwiseUnary<InT, OutT, VecSize, 1, 1, Cast>(&temp[0], &in_data[0],
                                                        Cast());
  kps::Reduce<OutT, VecSize, 1, 1, Add, Mode::kLocalMode>(&result, &temp[0],
                                                          Add(), true);
  kps::Reduce<OutT, 1, 1, 1, Add, Mode::kGlobalMode>(&result, &result, Add(),
                                                     true);
  if (store_fix == 0) {
    // first block's fix_size = 0;
    OutT tmp = static_cast<OutT>(0.0f);
    kps::WriteData<OutT, 1, 1, 1, true>(out + store_fix, &tmp, 1);
  }
  // store num of this block
  kps::WriteData<OutT, 1, 1, 1, true>(out + store_fix + 1, &result, 1);
}

template <typename InT, typename OutT, int VecSize>
__global__ void GetBlockCountKernel(const InT *in, OutT *out, int numel,
                                    int main_offset) {
  int data_offset = BLOCK_ID_X * BLOCK_NUM_X * VecSize;
  int stride = BLOCK_NUM_X * GRID_NUM_X * VecSize;
  int repeat = 0;
  for (; data_offset < main_offset; data_offset += stride) {
    GetBlockCountImpl<InT, OutT, VecSize, false>(in + data_offset, out,
                                                 BLOCK_NUM_X * VecSize, repeat);
    repeat++;  // to get the real blockIdx
  }

  int num = numel - main_offset;
  if (num > 0) {
    GetBlockCountImpl<InT, OutT, VecSize, true>(in + data_offset, out, num,
                                                repeat);
  }
}

/*
* Get block num prefix us one block, VecSize must be 2
* 1. Each thread load 2 data : threadIdx.x and threadIdx.x + blockDimx.x
* 2. Cumsum limitation is blockDim.x must be less than 512
*/

template <typename InT, typename OutT, typename Functor, int VecSize,
          bool IsBoundary>
__device__ void CumsumImpl(const InT *in, OutT *out, OutT *pre_cumsum, int num,
                           Functor func) {
  __shared__ OutT max_thread_data;
  OutT temp[VecSize];
  InT arg[VecSize];
  OutT result[VecSize];
  // init data_pr
  kps::Init<InT, VecSize>(&arg[0], static_cast<InT>(0.0f));
  // set pre_cumsum
  kps::Init<OutT, VecSize>(&temp[0], *pre_cumsum);
  // load data to arg
  kps::ReadData<InT, InT, VecSize, 1, 1, IsBoundary>(&arg[0], in, num, 1,
                                                     BLOCK_NUM_X, 1);
  // block cumsum
  kps::Cumsum<InT, OutT, 1, Functor>(&result[0], &arg[0], func);
  // result = cumsum_result + pre_cumsum
  kps::ElementwiseBinary<OutT, OutT, VecSize, 1, 1, Functor>(
      &result[0], &result[0], &temp[0], func);
  // get the last prefix sum
  if ((THREAD_ID_X == BLOCK_NUM_X - 1) && !IsBoundary) {
    max_thread_data = result[VecSize - 1];
  }
  __syncthreads();
  // update pre_cumsum
  *pre_cumsum = max_thread_data;
  kps::WriteData<OutT, OutT, VecSize, 1, 1, IsBoundary>(out, &result[0], num, 1,
                                                        BLOCK_NUM_X, 1);
}

template <typename InT, typename OutT, typename Functor, int VecSize>
__global__ void CumsumOneBlock(const InT *in, OutT *out, int numel,
                               int main_offset, Functor func) {
  int stride = BLOCK_NUM_X * VecSize;
  int offset = 0;
  OutT pre_cumsum = static_cast<OutT>(0);
  for (; offset < main_offset; offset += stride) {
    CumsumImpl<InT, OutT, Functor, VecSize, false>(
        in + offset, out + offset, &pre_cumsum, BLOCK_NUM_X * VecSize, func);
  }

  int num = numel - offset;
  if (num > 0) {
    CumsumImpl<InT, OutT, Functor, VecSize, true>(in + offset, out + offset,
                                                  &pre_cumsum, num, func);
  }
}
/**
* Get mask's index if mask == true
*/

template <typename InT, typename MT, typename OutT, typename Functor,
          int VecSize, int MaskData,
          int IsBoundary>  // SelectType = 1 Mask_select else where_index
__device__ void
SelectKernelImpl(OutT *out, const MT *mask, InT *in, Functor func, int num,
                 int data_offset, int store_rank) {
  const int kCVecSize = 2;
  // each thread cumsum 2 data
  using IdT = int64_t;
  // Set index data type
  using Add = kps::AddFunctor<IdT>;       // for cumsum
  using Cast = kps::IdentityFunctor<MT>;  // for mask

  IdT init_idx = static_cast<IdT>(0.0f);
  MT init_mask = static_cast<MT>(0.0f);

  IdT num_thread[kCVecSize];
  IdT cumsum_thread[kCVecSize];

  IdT index_reg[VecSize];
  OutT store_data[VecSize * framework::DDim::kMaxRank];
  InT in_data[VecSize];
  MT mask_data[VecSize];
  IdT mask_idt[VecSize];
  // init
  // init data_pr
  kps::Init<IdT, kCVecSize>(&cumsum_thread[0], init_idx);
  kps::Init<IdT, kCVecSize>(&num_thread[0], init_idx);
  kps::Init<MT, VecSize>(&mask_data[0], init_mask);
  // Load mask
  kps::ReadData<MT, VecSize, 1, 1, IsBoundary>(&mask_data[0], mask, num);
  // Cast from MT to int
  kps::ElementwiseUnary<MT, IdT, VecSize, 1, 1, Cast>(&mask_idt[0],
                                                      &mask_data[0], Cast());
  // Get the num of thread only num_thread[1] has data
  kps::Reduce<IdT, VecSize, 1, 1, Add, Mode::kLocalMode>(
      &num_thread[0], &mask_idt[0], Add(), true);
  // Get cumsum_thread cumsum from 0 to num_thread cumsum_thread[0] is the
  // thread_fix
  kps::Cumsum<IdT, IdT, 1, Add>(&cumsum_thread[0], &num_thread[0], Add());
  // Set data index of global
  kps::InitWithDataIndex<IdT, VecSize, 1, 1>(&index_reg[0], data_offset);
  // Get store data(index) according to mask_idt
  kps::OperatorTernary<MT, IdT, OutT, Functor>(&store_data[0], &mask_data[0],
                                               &index_reg[0], func, VecSize);
  // get thread_fix
  int thread_fix =
      (static_cast<int>(cumsum_thread[0] - num_thread[0]) * store_rank);
  // get how many data need to store
  int store_num = static_cast<int>(num_thread[0]) * store_rank;
  // thread store num data, each thread may has different num
  kps::details::WriteData<OutT>(out + thread_fix, &store_data[0], store_num);
}

template <typename MT, typename InT, typename OutT, typename Functor,
          int VecSize, int MaskData>
__global__ void SelectKernel(int64_t *out, const MT *mask, InT *in, InT *cumsum,
                             Functor func, const int64_t numel, int main_offset,
                             int store_rank) {
  int data_offset = BLOCK_ID_X * BLOCK_NUM_X * VecSize;
  int stride = BLOCK_NUM_X * GRID_NUM_X * VecSize;
  int repeat = 0;
  int size = VecSize * BLOCK_ID_X;
  for (; data_offset < main_offset; data_offset += stride) {
    // Cumsum index
    int idx_cumsum = repeat * GRID_NUM_X + BLOCK_ID_X;
    // niuliling todo: us ReadData API
    int block_store_offset = cumsum[idx_cumsum];
    SelectKernelImpl<InT, MT, OutT, Functor, VecSize, MaskData, false>(
        out + block_store_offset * store_rank, mask + data_offset,
        in + data_offset, func, size, data_offset, store_rank);
    repeat++;
  }

  int num = numel - data_offset;
  if (num > 0) {
    // Cumsum index
    int idx_cumsum = repeat * GRID_NUM_X + BLOCK_ID_X;
    // niuliling todo: us ReadData API
    int block_store_offset = static_cast<int>(cumsum[idx_cumsum]);
    SelectKernelImpl<InT, MT, OutT, Functor, VecSize, MaskData, true>(
        out + block_store_offset * store_rank, mask + data_offset,
        in + data_offset, func, num, data_offset, store_rank);
  }
}

template <typename T1, typename T2, typename OutT>
struct IndexFunctor {
  T2 stride[paddle::framework::DDim::kMaxRank];
  int dims;
  HOSTDEVICE IndexFunctor(const framework::DDim &in_dims) {
    dims = in_dims.size();
    std::vector<T2> strides_in_tmp;
    strides_in_tmp.resize(dims, 1);
    // get strides according to in_dims
    for (T2 i = 1; i < dims; i++) {
      strides_in_tmp[i] = strides_in_tmp[i - 1] * in_dims[dims - i];
    }
    memcpy(stride, strides_in_tmp.data(), dims * sizeof(T2));
  }

  HOSTDEVICE inline void operator()(OutT *out, const T1 *mask, const T2 *index,
                                    const int num) {
    int store_fix = 0;
    for (int idx = 0; idx < num; idx++) {
      if (mask[idx]) {
        T2 data_index = index[idx];
        // get index
        for (int rank_id = dims - 1; rank_id >= 0; --rank_id) {
          out[store_fix] = static_cast<OutT>(data_index / stride[rank_id]);
          data_index = data_index % stride[rank_id];
          store_fix++;
        }
      }
    }
  }
};

template <typename T>
class CUDAWhereIndexKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *condition = context.Input<framework::Tensor>("Condition");
    auto *out = context.Output<framework::Tensor>("Out");
    auto &dev_ctx = context.template device_context<CUDADeviceContext>();

    auto stream = dev_ctx.stream();
    const T *cond_data = condition->data<T>();
    const int64_t numel = condition->numel();
    auto dims = condition->dims();
    int rank = dims.size();
    // alloc for cpu
    auto h_array_mem =
        memory::Alloc(platform::CPUPlace(), (rank + 1) * sizeof(int64_t));
    int64_t *h_stride_array = reinterpret_cast<int64_t *>(h_array_mem->ptr());
    // calculate the inclusive prefix sum of "true_num_array"
    // to get the index of "out" tensor,
    // and the total number of cond_data[i]==true.
    // Example:
    // condition: F T T F F F T T
    // before:    0 1 1 0 0 0 1 1
    // after:     0 1 2 2 2 2 3 4
    // out:       1 2 6 7
    // 1.1 get stored data num of per block
    const int VecSize = 4;
    int block = 256;
    int num_per_block = VecSize * block;
    const int64_t need_grids = (numel + num_per_block - 1) / num_per_block;
    const int grid = std::min(need_grids, static_cast<int64_t>(256));
    int main_offset = (numel) / num_per_block * num_per_block;
    // 1.2 alloc tmp data for CoutBlock
    int size_count_block = need_grids + 1;
    auto count_mem = memory::Alloc(dev_ctx, size_count_block * sizeof(int64_t));
    int64_t *count_data = reinterpret_cast<int64_t *>(count_mem->ptr());
    // 1.3 launch CountKernl
    GetBlockCountKernel<T, int64_t, VecSize><<<grid, block, 0, stream>>>(
        cond_data, count_data, numel, main_offset);
    memory::Copy(platform::CPUPlace(), h_stride_array, dev_ctx.GetPlace(),
                 count_data, sizeof(int64_t), dev_ctx.stream());
    // 2.1 alloc cumsum data for CoutBlock prefix
    auto cumsum_mem =
        memory::Alloc(dev_ctx, size_count_block * sizeof(int64_t));
    int64_t *cumsum_data = reinterpret_cast<int64_t *>(cumsum_mem->ptr());
    // 2.2 get prefix of count_data for real out_index
    int block_c = 256;
    int main_offset_c = size_count_block / (2 * block_c) * (2 * block_c);
    CumsumOneBlock<int64_t, int64_t, kps::AddFunctor<int64_t>,
                   2><<<1, block_c, 0, stream>>>(
        count_data, cumsum_data, size_count_block, main_offset_c,
        kps::AddFunctor<int64_t>());
    // 3.1 set temp ptr for in;
    // 3.1 alloc for out
    // 3.1.1 get true_num for gpu place the last cumsum is the true_num
    memory::Copy(platform::CPUPlace(), h_stride_array + 1, dev_ctx.GetPlace(),
                 cumsum_data + need_grids, sizeof(int64_t), dev_ctx.stream());
    // 3.1.2 allock for out with total_true_num
    out->Resize(
        phi::make_ddim({static_cast<int64_t>(h_stride_array[1]), rank}));
    auto out_data = out->mutable_data<int64_t>(context.GetPlace());
    // 3.2 get true data's index according to cond_data and cumsum_data
    int64_t *tmp_in = nullptr;
    using Functor = IndexFunctor<T, int64_t, int64_t>;
    Functor index_functor = Functor(dims);
    SelectKernel<T, int64_t, int64_t, Functor, VecSize,
                 0><<<grid, block, 0, stream>>>(out_data, cond_data, tmp_in,
                                                cumsum_data, index_functor,
                                                numel, main_offset, rank);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(where_index, ops::CUDAWhereIndexKernel<int64_t>,
                        ops::CUDAWhereIndexKernel<int>,
                        ops::CUDAWhereIndexKernel<int16_t>,
                        ops::CUDAWhereIndexKernel<bool>,
                        ops::CUDAWhereIndexKernel<float>,
                        ops::CUDAWhereIndexKernel<double>);
