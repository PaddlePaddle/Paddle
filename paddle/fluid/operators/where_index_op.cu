/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include <algorithm>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/where_index_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/primitive/kernel_primitives.h"
#include "paddle/phi/kernels/select_impl.cu.h"

namespace paddle {
namespace operators {
template <typename T1, typename T2, typename OutT>
struct IndexFunctor {
  T2 stride[paddle::framework::DDim::kMaxRank];
  int dims;
  explicit IndexFunctor(const framework::DDim &in_dims) {
    dims = in_dims.size();
    std::vector<T2> strides_in_tmp;
    strides_in_tmp.resize(dims, 1);
    // get strides according to in_dims
    for (T2 i = 1; i < dims; i++) {
      strides_in_tmp[i] = strides_in_tmp[i - 1] * in_dims[dims - i];
    }
    memcpy(stride, strides_in_tmp.data(), dims * sizeof(T2));
  }

  HOSTDEVICE inline void operator()(OutT *out, const T1 *mask, const T2 *index,
                                    const int num) {
    int store_fix = 0;
    for (int idx = 0; idx < num; idx++) {
      if (mask[idx]) {
        T2 data_index = index[idx];
        // get index
        for (int rank_id = dims - 1; rank_id >= 0; --rank_id) {
          out[store_fix] = static_cast<OutT>(data_index / stride[rank_id]);
          data_index = data_index % stride[rank_id];
          store_fix++;
        }
      }
    }
  }
};

template <typename T>
class CUDAWhereIndexKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *condition = context.Input<framework::Tensor>("Condition");
    auto *out = context.Output<framework::Tensor>("Out");
    using CUDADeviceContext = paddle::platform::CUDADeviceContext;
    auto &dev_ctx = context.template device_context<CUDADeviceContext>();
    framework::Tensor in_data;
    auto dims = condition->dims();
    using Functor = IndexFunctor<T, int64_t, int64_t>;
    Functor index_functor = Functor(dims);
    phi::SelectKernel<T, T, int64_t, 0, Functor>(
        static_cast<const phi::GPUContext &>(dev_ctx), *condition, in_data, out,
        index_functor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(where_index, ops::CUDAWhereIndexKernel<int64_t>,
                        ops::CUDAWhereIndexKernel<int>,
                        ops::CUDAWhereIndexKernel<int16_t>,
                        ops::CUDAWhereIndexKernel<bool>,
                        ops::CUDAWhereIndexKernel<float>,
                        ops::CUDAWhereIndexKernel<double>);
