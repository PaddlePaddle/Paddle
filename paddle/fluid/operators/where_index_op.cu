#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include <algorithm>
#include "paddle/fluid/framework/ddim.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/where_index_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

using CUDADeviceContext = paddle::platform::CUDADeviceContext;

template <typename T>
__global__ void GetTrueNum(const T *cond_data, const int64_t numel,
                           int64_t *true_num_array) {
  const int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int64_t idx = tid; idx < numel; idx += gridDim.x * blockDim.x) {
    true_num_array[idx] =
        static_cast<int64_t>(static_cast<bool>(cond_data[idx]));
  }
}

template <typename T>
__global__ void SetTrueIndex(int64_t *out_ptr, const T *cond_data,
                             const int64_t numel, const int64_t *stride_array,
                             const int64_t rank,
                             const int64_t *true_num_array) {
  const int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int64_t idx = tid; idx < numel; idx += gridDim.x * blockDim.x) {
    // true_num_array is calculated by hipcub::InclusiveSum,
    // cause the first element of true_num_array is 1,
    // so we need substract 1 to get true index.
    const int64_t true_index = true_num_array[idx] - 1;
    if (static_cast<bool>(cond_data[idx])) {
      int64_t rank_index = idx;
      for (int j = 0; j < rank; j++) {
        const int64_t out_index = rank_index / stride_array[j];
        out_ptr[true_index * rank + j] = out_index;
        rank_index -= out_index * stride_array[j];
      }
    }
  }
}

template <typename T>
class CUDAWhereIndexKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *condition = context.Input<framework::Tensor>("Condition");
    auto *out = context.Output<framework::Tensor>("Out");
    auto &dev_ctx = context.template device_context<CUDADeviceContext>();

    const T *cond_data = condition->data<T>();
    const int64_t numel = condition->numel();
    auto dims = condition->dims();
    const int rank = dims.size();

    auto d_array_mem = memory::Alloc(dev_ctx, (numel + rank) * sizeof(int64_t));
    auto h_array_mem =
        memory::Alloc(platform::CPUPlace(), (rank + 1) * sizeof(int64_t));

    // "stride_array" is an array and len(stride_array)==rank,
    // each element is the stride of each dimension -- the length from i to i+1.
    int64_t *h_stride_array = reinterpret_cast<int64_t *>(h_array_mem->ptr());
    int64_t *d_stride_array = reinterpret_cast<int64_t *>(d_array_mem->ptr());

    // "true_num_array" is an array and len(stride_array)==numel,
    // at the beginning,
    // "true_num_array" will set 1 if condition[i] == true else 0,
    // then it will be calculated by hipcub::InclusiveSum,
    // so that we can get the true number before i as the out index
    int64_t *d_true_num_array = d_stride_array + rank;

    // the total_true_num is the total number of condition[i] == true
    int64_t *h_total_true_num = h_stride_array + rank;

    // alloce cub memory
    size_t cub_size = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, cub_size, d_true_num_array,
                                  d_true_num_array, numel, dev_ctx.stream());
    auto cub_mem = memory::Alloc(dev_ctx, cub_size * sizeof(int64_t));
    void *cub_data = cub_mem->ptr();

    // set d_true_num_array[i]=1 if cond_data[i]==true else 0
    const int threads = std::min(numel, static_cast<int64_t>(128));
    const int64_t need_grids = (numel + threads - 1) / threads;
    const int grids = std::min(need_grids, static_cast<int64_t>(256));
    GetTrueNum<T><<<grids, threads, 0, dev_ctx.stream()>>>(cond_data, numel,
                                                           d_true_num_array);

    // calculate the inclusive prefix sum of "true_num_array"
    // to get the index of "out" tensor,
    // and the total number of cond_data[i]==true.
    // Example:
    // condition: F T T F F F T T
    // before:    0 1 1 0 0 0 1 1
    // after:     0 1 2 2 2 2 3 4
    // out:       1 2 6 7
    hipcub::DeviceScan::InclusiveSum(cub_data, cub_size, d_true_num_array,
                                  d_true_num_array, numel, dev_ctx.stream());

    // calculate each dimension's stride
    h_stride_array[rank - 1] = 1;
    for (int i = rank - 2; i >= 0; i--) {
      h_stride_array[i] = h_stride_array[i + 1] * dims[i + 1];
    }
    memory::Copy(BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                 d_stride_array, platform::CPUPlace(), h_stride_array,
                 rank * sizeof(int64_t), dev_ctx.stream());

    // get total ture number and set output size
    // the last element of hipcub::InclusiveSum is the total number
    memory::Copy(platform::CPUPlace(), h_total_true_num,
                 BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                 d_true_num_array + numel - 1, sizeof(int64_t),
                 dev_ctx.stream());
    dev_ctx.Wait();

    int64_t true_num = *h_total_true_num;
    out->Resize(framework::make_ddim({static_cast<int64_t>(true_num), rank}));
    auto out_data = out->mutable_data<int64_t>(context.GetPlace());

    if (true_num == 0) {
      return;
    }

    // using true_num_array and stride_array to calculate the output index
    SetTrueIndex<T><<<grids, threads, 0, dev_ctx.stream()>>>(
        out_data, cond_data, numel, d_stride_array, rank, d_true_num_array);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(where_index, ops::CUDAWhereIndexKernel<int64_t>,
                        ops::CUDAWhereIndexKernel<int>,
                        ops::CUDAWhereIndexKernel<bool>,
                        ops::CUDAWhereIndexKernel<float>,
                        ops::CUDAWhereIndexKernel<double>);
