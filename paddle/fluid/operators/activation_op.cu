#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

#define REGISTER_ACTIVATION_CUDA_KERNEL(act_type, op_name, functor,         \
                                        grad_functor)                       \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type,                                                             \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<float>>,  \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<double>>, \
      ops::ActivationKernel<plat::CUDADeviceContext,                        \
                            ops::functor<plat::float16>>);                  \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type##_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,   \
                                                 ops::grad_functor<float>>, \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<double>>,                 \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<plat::float16>>);

FOR_EACH_ACTIVATION_OP(REGISTER_ACTIVATION_CUDA_KERNEL);

/* ======================== leaky relu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(leaky_relu, LeakyRelu, LeakyReluFunctor,
                                LeakyReluGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    leaky_relu_grad_grad,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<
        plat::CUDADeviceContext, ops::LeakyReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================    relu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(relu, Relu, ReluFunctor, ReluGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    relu_grad_grad,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   sqrt register  ============================= */
REGISTER_ACTIVATION_CUDA_KERNEL(sqrt, Sqrt, SqrtFunctor, SqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    sqrt_grad_grad,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<float>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<double>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================  square register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(square, Square, SquareFunctor,
                                SquareGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    square_grad_grad,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<float>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<double>>,
    ops::SquareDoubleGradKernel<plat::CUDADeviceContext,
                                ops::SquareGradGradFunctor<plat::float16>>);
/* ========================================================================== */
