#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using float16 = paddle::platform::float16;

template <typename T>
struct CudaVecType {
  using type = T;
  static constexpr int vecsize = 1;
};

template <>
struct CudaVecType<platform::float16> {
  using type = __half2;
  static constexpr int vecsize = 2;
};

template <>
struct CudaVecType<float> {
  using type = float4;
  static constexpr int vecsize = 4;
};

template <typename T>
class BaseGPUFunctor {
 public:
  using ELEMENT_TYPE = T;

  using AttrPair = std::vector<std::pair<const char*, float*>>;

  AttrPair GetAttrs() { return AttrPair(); }
};

/* ========================================================================== */

/* ===========================    relu forward   ============================ */
template <typename T>
class ReluGPUFunctor : public BaseGPUFunctor<T> {
 private:
  T zero_;

 public:
  ReluGPUFunctor() { zero_ = static_cast<T>(0.0f); }

  // for relu forward when T is double
  __device__ __forceinline__ typename CudaVecType<T>::type Compute(
      const typename CudaVecType<T>::type in) {
    // relu forward : out = max(x, 0)
    return in > zero_ ? in : zero_;
  }

  // when num % vecsize != 0 this func will be used
  __device__ __forceinline__ T ComputeRemainder(const T in) {
    // relu forward : out = max(x, 0)
    return in > zero_ ? in : zero_;
  }
};

template <>
__device__ __forceinline__ CudaVecType<float>::type
ReluGPUFunctor<float>::Compute(const CudaVecType<float>::type in) {
  // relu forward : out = max(in, 0)
  return make_float4((in.x > zero_) * (in.x), (in.y > zero_) * (in.y),
                     (in.z > zero_) * (in.z), (in.w > zero_) * (in.w));
}

template <>
__device__ __forceinline__ CudaVecType<float16>::type
ReluGPUFunctor<float16>::Compute(const CudaVecType<float16>::type in) {
// relu forward : out = max(in, 0)
#ifdef __HIPCC__ || CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const half2 kzero = __float2half2_rn(0.0f);
  return __hmul2(__hgt2(in, kzero), in);
#else
  const float2 xx = __half22float2(in);
  return __floats2half2_rn((xx.x > 0.0f) * static_cast<float>(xx.x),
                           (xx.y > 0.0f) * static_cast<float>(xx.y));
#endif
}
/* ========================================================================== */

/* ===========================    relu backward   ============================
 */

template <typename T>
class ReluGradGPUFunctor : public BaseGPUFunctor<T> {
 private:
  T zero_;

 public:
  ReluGradGPUFunctor() { zero_ = static_cast<T>(0.0f); }

  // for relu backward when T is double
  __device__ __forceinline__ typename CudaVecType<T>::type Compute(
      const typename CudaVecType<T>::type out,
      const typename CudaVecType<T>::type dout) {
    return out > zero_ ? dout : zero_;
  }

  // when num % vecsize != 0 this func will be used
  __device__ __forceinline__ T ComputeRemainder(const T out, const T dout) {
    // relu backward : dx = out > 0 ? dout : 0
    return out > zero_ ? dout : zero_;
  }

  static constexpr ActBwdOpFwdDeps FwdDeps() { return kDepOut; }
};

template <>
__device__ __forceinline__ CudaVecType<float>::type
ReluGradGPUFunctor<float>::Compute(const CudaVecType<float>::type out,
                                   const CudaVecType<float>::type dout) {
  // relu backward : dx = out > 0 ? dout : 0;
  return make_float4((out.x > zero_) * (dout.x), (out.y > zero_) * (dout.y),
                     (out.z > zero_) * (dout.z), (out.w > zero_) * (dout.w));
}

template <>
__device__ __forceinline__ CudaVecType<float16>::type
ReluGradGPUFunctor<float16>::Compute(const CudaVecType<float16>::type out,
                                     const CudaVecType<float16>::type dout) {
// relu backward : dx = out > 0 ? dout : 0;
#ifdef __HIPCC__ || CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const half2 kzero = __float2half2_rn(0.0f);
  return __hmul2(__hgt2(out, kzero), dout);
#else
  const float2 xx = __half22float2(out);
  const float2 yy = __half22float2(dout);
  return __floats2half2_rn((xx.x > 0.0f) * static_cast<float>(yy.x),
                           (xx.y > 0.0f) * static_cast<float>(yy.y));
#endif
}

/* ========================================================================== */
/* ========================    leaky relu forward    ========================
 */
template <typename T>
class LeakyReluGPUFunctor : public BaseGPUFunctor<T> {
 private:
  T zero_;
  float alpha_;

 public:
  LeakyReluGPUFunctor() { zero_ = static_cast<T>(0.0f); }

  typename BaseActivationFunctor<T>::AttrPair GetAttrs() {
    return {{"alpha", &alpha_}};
  }
  // leakyrelu forward : out = x > 0 ? x : x * alpha
  __device__ __forceinline__ typename CudaVecType<T>::type Compute(
      const typename CudaVecType<T>::type in) {
    return in > zero_ ? in : static_cast<T>(alpha_) * in;
  }

  __device__ __forceinline__ T ComputeRemainder(const T in) {
    // leakyrelu forward : out = x > 0 ? x : x * alpha
    return in > zero_ ? in : static_cast<T>(alpha_) * in;
  }
};

template <>
__device__ __forceinline__ CudaVecType<float>::type
LeakyReluGPUFunctor<float>::Compute(const CudaVecType<float>::type in) {
  // leakyrelu forward : out = x > 0 ? x : x * alpha
  return make_float4((in.x > zero_) ? (in.x) : (in.x) * alpha_,
                     (in.y > zero_) ? (in.y) : (in.y) * alpha_,
                     (in.z > zero_) ? (in.z) : (in.z) * alpha_,
                     (in.w > zero_) ? (in.w) : (in.w) * alpha_);
}

template <>
__device__ __forceinline__ CudaVecType<float16>::type
LeakyReluGPUFunctor<float16>::Compute(const CudaVecType<float16>::type in) {
  // leakyrelu forward : out = x > 0 ? x : x * alpha
  const float2 xx = __half22float2(in);
  return __floats2half2_rn((xx.x > 0.0f) ? xx.x : xx.x * alpha_,
                           (xx.y > 0.0f) ? xx.y : xx.y * alpha_);
}
/* ========================================================================== */

/* ===========================  leaky relu backward   =======================
 */
template <typename T>
class LeakyReluGradGPUFunctor : public BaseGPUFunctor<T> {
 private:
  T zero_;
  float alpha_;

 public:
  LeakyReluGradGPUFunctor() { zero_ = static_cast<T>(0.0f); }

  typename BaseActivationFunctor<T>::AttrPair GetAttrs() {
    return {{"alpha", &alpha_}};
  }

  // for leaky relu backward when T is double
  __device__ __forceinline__ typename CudaVecType<T>::type Compute(
      const typename CudaVecType<T>::type in,
      const typename CudaVecType<T>::type dout) {
    // leakyrelu backward : dx = x > 0 ? dout : alpha * dout
    return in > zero_ ? dout : static_cast<T>(alpha_) * dout;
  }

  // when num % vecsize != 0 this func will be used
  __device__ __forceinline__ T ComputeRemainder(const T in, const T dout) {
    // leakyrelu backward : dx = x > 0 ? dout : alpha * dout
    return in > zero_ ? dout : static_cast<T>(alpha_) * dout;
  }

  static constexpr ActBwdOpFwdDeps FwdDeps() { return kDepX; }
};

template <>
__device__ __forceinline__ CudaVecType<float>::type
LeakyReluGradGPUFunctor<float>::Compute(const CudaVecType<float>::type in,
                                        const CudaVecType<float>::type dout) {
  // leakyrelu backward : dx = x > 0 ? dout : alpha * dout
  return make_float4((in.x > zero_) ? (dout.x) : alpha_ * (dout.x),
                     (in.y > zero_) ? (dout.y) : alpha_ * (dout.y),
                     (in.z > zero_) ? (dout.z) : alpha_ * (dout.z),
                     (in.w > zero_) ? (dout.w) : alpha_ * (dout.w));
}

template <>
__device__ __forceinline__ CudaVecType<float16>::type LeakyReluGradGPUFunctor<
    float16>::Compute(const CudaVecType<float16>::type in,
                      const CudaVecType<float16>::type dout) {
  // leakyrelu backward : dx = x > 0 ? dout : alpha * dout
  const float2 xx = __half22float2(in);
  const float2 yy = __half22float2(dout);
  return __floats2half2_rn((xx.x > 0.0f) ? yy.x : alpha_ * yy.x,
                           (xx.y > 0.0f) ? yy.y : alpha_ * yy.y);
}

/* ========================================================================== */

template <typename T, typename Functor>
__global__ void ActivationGradKernelVec(const T* forward_data, const T* dout,
                                        T* dx, int num, Functor functor) {
  using VecType = typename CudaVecType<T>::type;
  constexpr int vecsize = CudaVecType<T>::vecsize;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  int loop = num / vecsize;
  int tail = num % vecsize;
  const VecType* in_forward = reinterpret_cast<const VecType*>(forward_data);
  const VecType* in_dout = reinterpret_cast<const VecType*>(dout);
  VecType* out = reinterpret_cast<VecType*>(dx);
  VecType forward_vec, dout_vec;
  T in_data, dout_data;
  for (int i = idx; i < loop; i += stride) {
#ifdef __HIPCC__ || __CUDA_ARCH__ >= 350
    forward_vec = __ldg(in_forward + i);
    dout_vec = __ldg(in_dout + i);
#else
    forward_vec = in_forward[i];
    dout_vec = in_dout[i];
#endif
    out[i] = functor.Compute(forward_vec, dout_vec);
  }

  while (idx == loop && tail) {
    in_data = forward_data[num - tail];
    dout_data = dout[num - tail];
    dx[num - tail] = functor.ComputeRemainder(in_data, dout_data);
    --tail;
  }
}

template <typename T, typename Functor>
__global__ void ActivationkernelVec(const T* src, T* dst, int num,
                                    Functor functor) {
  constexpr int vecsize = CudaVecType<T>::vecsize;
  using VecType = typename CudaVecType<T>::type;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  int loop = num / vecsize;
  int tail = num % vecsize;
  const VecType* in = reinterpret_cast<const VecType*>(src);
  VecType* out = reinterpret_cast<VecType*>(dst);
  VecType x_vec;
  for (int i = idx; i < loop; i += stride) {
#ifdef __HIPCC__ || __CUDA_ARCH__ >= 350
    x_vec = __ldg(in + i);
#else
    x_vec = in[i];
#endif
    out[i] = functor.Compute(x_vec);
  }

  while (idx == loop && tail) {
    dst[num - tail] = functor.ComputeRemainder(src[num - tail]);
    --tail;
  }
}

template <typename DeviceContext, typename Functor>
class ActivationGPUKernel
    : public framework::OpKernel<typename Functor::ELEMENT_TYPE> {
 public:
  using T = typename Functor::ELEMENT_TYPE;
  void Compute(const framework::ExecutionContext& context) const override {
    const framework::Tensor* in_x = nullptr;
    framework::Tensor* out = nullptr;
    ExtractActivationTensor(context, &in_x, &out);
    auto& dev_ctx = context.template device_context<DeviceContext>();

    int num = in_x->numel();
    const T* input_data = in_x->data<T>();
    T* output_data = out->mutable_data<T>(dev_ctx.GetPlace(),
                                          static_cast<size_t>(num * sizeof(T)));

    int block = 512;
#ifdef __HIPCC__
    block = 256;
#endif
    Functor functor;
    auto attrs = functor.GetAttrs();
    for (auto& attr : attrs) {
      *attr.second = context.Attr<float>(attr.first);
    }
    constexpr int vecsize = CudaVecType<T>::vecsize;
    int grid = max((num / vecsize + block - 1) / block, 1);
    auto stream = context.cuda_device_context().stream();
    ActivationkernelVec<T, Functor><<<grid, block, 0, stream>>>(
        input_data, output_data, num, functor);
  }
};

template <typename DeviceContext, typename Functor>
class ActivationGradGPUKernel
    : public framework::OpKernel<typename Functor::ELEMENT_TYPE> {
 public:
  using T = typename Functor::ELEMENT_TYPE;
  void Compute(const framework::ExecutionContext& context) const override {
    const framework::Tensor *x, *out, *d_out;
    framework::Tensor* d_x = nullptr;
    x = out = d_out = nullptr;
    ExtractActivationGradTensor<Functor::FwdDeps()>(context, &x, &out, &d_out,
                                                    &d_x);
    int numel = d_out->numel();
    auto& dev_ctx = context.template device_context<DeviceContext>();
    auto* dx_data = d_x->mutable_data<T>(
        dev_ctx.GetPlace(), static_cast<size_t>(numel * sizeof(T)));
    auto* dout_data = d_out->data<T>();

    auto* forward_data = dout_data;
    if (static_cast<int>(Functor::FwdDeps()) == static_cast<int>(kDepOut)) {
      // Only need forward output Out
      forward_data = out->data<T>();
    } else if (static_cast<int>(Functor::FwdDeps()) ==
               static_cast<int>(kDepX)) {
      // Only need forward input X
      forward_data = x->data<T>();
    }

    int block = 512;
#ifdef __HIPCC__
    block = 256;
#endif

    Functor functor;
    auto attrs = functor.GetAttrs();
    for (auto& attr : attrs) {
      *attr.second = context.Attr<float>(attr.first);
    }
    constexpr int vecsize = CudaVecType<T>::vecsize;
    int grid = max((numel / vecsize + block - 1) / block, 1);
    auto stream = context.cuda_device_context().stream();
    ActivationGradKernelVec<T, Functor><<<grid, block, 0, stream>>>(
        forward_data, dout_data, dx_data, numel, functor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

#define REGISTER_ACTIVATION_CUDA_KERNEL(act_type, op_name, functor,         \
                                        grad_functor)                       \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type,                                                             \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<float>>,  \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<double>>, \
      ops::ActivationKernel<plat::CUDADeviceContext,                        \
                            ops::functor<plat::float16>>);                  \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type##_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,   \
                                                 ops::grad_functor<float>>, \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<double>>,                 \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<plat::float16>>);
FOR_EACH_ACTIVATION_OP(REGISTER_ACTIVATION_CUDA_KERNEL);

#define REGISTER_ACTIVATION_GPU_KERNEL(act_type, op_name, functor,             \
                                       grad_functor)                           \
  REGISTER_OP_CUDA_KERNEL(                                                     \
      act_type, ops::ActivationGPUKernel<paddle::platform::CUDADeviceContext,  \
                                         ops::functor<float>>,                 \
      ops::ActivationGPUKernel<paddle::platform::CUDADeviceContext,            \
                               ops::functor<double>>,                          \
      ops::ActivationGPUKernel<plat::CUDADeviceContext,                        \
                               ops::functor<plat::float16>>);                  \
  REGISTER_OP_CUDA_KERNEL(                                                     \
      act_type##_grad, ops::ActivationGradGPUKernel<plat::CUDADeviceContext,   \
                                                    ops::grad_functor<float>>, \
      ops::ActivationGradGPUKernel<plat::CUDADeviceContext,                    \
                                   ops::grad_functor<double>>,                 \
      ops::ActivationGradGPUKernel<plat::CUDADeviceContext,                    \
                                   ops::grad_functor<plat::float16>>);

/* ======================== leaky relu register  ============================ */
REGISTER_ACTIVATION_GPU_KERNEL(leaky_relu, LeakyRelu, LeakyReluGPUFunctor,
                               LeakyReluGradGPUFunctor);

REGISTER_OP_CUDA_KERNEL(
    leaky_relu_grad_grad,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<
        plat::CUDADeviceContext, ops::LeakyReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ======================== elu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(elu, ELU, ELUFunctor, ELUGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    elu_grad_grad, ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                                            ops::ELUGradGradFunctor<float>>,
    ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                             ops::ELUGradGradFunctor<double>>,
    ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                             ops::ELUGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================    relu register  ============================ */
REGISTER_ACTIVATION_GPU_KERNEL(relu, Relu, ReluGPUFunctor, ReluGradGPUFunctor);

REGISTER_OP_CUDA_KERNEL(
    relu_grad_grad,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================    tanh register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(tanh, Tanh, TanhFunctor, TanhGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    tanh_grad_grad,
    ops::TanhDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::TanhGradGradFunctor<float>>,
    ops::TanhDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::TanhGradGradFunctor<double>>,
    ops::TanhDoubleGradKernel<plat::CUDADeviceContext,
                              ops::TanhGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   sqrt register  ============================= */
REGISTER_ACTIVATION_CUDA_KERNEL(sqrt, Sqrt, SqrtFunctor, SqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    sqrt_grad_grad,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<float>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<double>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   rsqrt register  =============================
 */
REGISTER_ACTIVATION_CUDA_KERNEL(rsqrt, Rsqrt, RsqrtFunctor, RsqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    rsqrt_grad_grad,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<float>>,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<double>>,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================  square register  ============================ */
REGISTER_OP_CUDA_KERNEL(
    square,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<float>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<double>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<int>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<int64_t>>,
    ops::ActivationKernel<plat::CUDADeviceContext,
                          ops::SquareFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    square_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                           ops::SquareGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<int>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<int64_t>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<plat::float16>>);

REGISTER_OP_CUDA_KERNEL(
    square_grad_grad,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<float>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<double>>,
    ops::SquareDoubleGradKernel<plat::CUDADeviceContext,
                                ops::SquareGradGradFunctor<plat::float16>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<int>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<int64_t>>);
/* ========================================================================== */

/* ==========================   pow register  ============================ */

REGISTER_OP_CUDA_KERNEL(
    pow, ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<float>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<double>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<int>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<int64_t>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    pow_grad,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<float>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<double>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<int>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<int64_t>>,
    ops::PowGradKernel<plat::CUDADeviceContext,
                       ops::PowGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ==========================   exp register  ============================ */

REGISTER_OP_CUDA_KERNEL(
    exp, ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<float>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<double>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<int>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<int64_t>>,
    ops::ActivationKernel<plat::CUDADeviceContext,
                          ops::ExpFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    exp_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                        ops::ExpGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<int>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<int64_t>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ==========================  Log register ==================================*/
REGISTER_ACTIVATION_CUDA_KERNEL(log, Log, LogFunctor, LogGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    log_grad_grad, ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                                            ops::LogGradGradFunctor<float>>,
    ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                             ops::LogGradGradFunctor<double>>,
    ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                             ops::LogGradGradFunctor<plat::float16>>);
/* ========================================================================== */
