#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
template <typename T>
__global__ void reluKernelCudaHalf2(const T* in, T* out, int num) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int loop = num >> 1;
  int stride = blockDim.x * gridDim.x;
  const __half2* src = reinterpret_cast<const __half2*>(in);
  __half2* dst = reinterpret_cast<__half2*>(out);
  const half2 kzero = __float2half2_rn(0.0f);
  for (int i = idx; i < loop; i += stride) {
#if __CUDA_ARCH__ >= 530 || CUDA_VERSION >= 300
    dst[i] = __hmul2(__hgt2(__ldg(src + i), kzero), __ldg(src + i));
#else
    const float2 xx = __halfi22float2(src[i]);
    dst[i] = __floats2half2_rn(xx.x > 0.0f ? static_cast<float>(xx.x) : 0.0f,
                               xx.y > 0.0f ? static_cast<float>(xx.y) : 0.0f);
#endif
  }

  if (idx == loop && num % 2 == 1) {
    T zero = (T)(0.0f);
    out[num - 1] = in[num - 1] > zero ? in[num - 1] : zero;
  }
}
template <typename T>
__global__ void reluKernelCudaFloat4(const T* in, T* out, int num) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const float4* src = reinterpret_cast<const float4*>(in);
  float4* dst = reinterpret_cast<float4*>(out);
  float4 temp;
  int loop = num >> 2;
  for (int i = idx; i < loop; i += blockDim.x * gridDim.x) {
    temp = src[i];
    temp.x = max(temp.x, 0.0f);
    temp.y = max(temp.y, 0.0f);
    temp.z = max(temp.z, 0.0f);
    temp.w = max(temp.w, 0.0f);
    dst[i] = temp;
  }
  int tail = num % 4;
  while (idx == loop && tail) {
    temp.x = static_cast<float>(in[num - tail]);
    out[num - tail] = max(temp.x, 0.0f);
    --tail;
  }
}
template <typename T>
__global__ void reluKernelCudaDouble(const T* src, T* dst, int num) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  T zero = (T)(0.0f);

  for (int idx = i; idx < num; idx += gridDim.x * blockDim.x) {
    dst[i] = src[idx] > zero ? src[idx] : zero;
  }
}
template <typename T, int vec>
struct ReluGPUFunctor : public BaseActivationFunctor<T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& in, framework::Tensor* out,
                  int num) {
    const T* input_data = in.data<T>();
    T* output_data = out->mutable_data<T>(context.GetPlace());
    int block = 512;
    int grid = (num + block - 1) / block;
    int grid2 = (num / 2 + block - 1) / block;
    int grid4 = (num / 4 + block - 1) / block;
    switch (vec) {
      case 1:  // float16 -> half2
        reluKernelCudaHalf2<<<grid2, block>>>(input_data, output_data, num);
        break;
      case 2:  // float -> float4
        reluKernelCudaFloat4<<<grid4, block>>>(input_data, output_data, num);
        break;
      case 4:  // double -> double
        reluKernelCudaDouble<<<grid, block>>>(input_data, output_data, num);
        break;
      default:
        break;
    }
  }
};
template <typename DeviceContext, typename Functor>
class ReluBaseKernel
    : public framework::OpKernel<typename Functor::ELEMENT_TYPE> {
 public:
  using T = typename Functor::ELEMENT_TYPE;
  void Compute(const framework::ExecutionContext& context) const override {
    const Tensor* in_x = context.Input<Tensor>("X");
    Tensor* out = context.Output<Tensor>("Out");
    auto& dev_ctx = context.template device_context<DeviceContext>();
    int num = in_x->numel();
    Functor functor;
    functor(dev_ctx, *in_x, out, num);
  }
};

template <typename DeviceContext, typename Functor>
class ReluGradKernel
    : public framework::OpKernel<typename Functor::ELEMENT_TYPE> {
 public:
  using T = typename Functor::ELEMENT_TYPE;
  void Compute(const framework::ExecutionContext& context) const override {
    const Tensor* in_x = context.Input<Tensor>("X");
    Tensor* out = context.Output<Tensor>("Out");
    auto& dev_ctx = context.template device_context<DeviceContext>();
    int num = in_x->numel();
    Functor functor;
    functor(dev_ctx, *in_x, out, num);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

#define REGISTER_ACTIVATION_CUDA_KERNEL(act_type, op_name, functor,         \
                                        grad_functor)                       \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type,                                                             \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<float>>,  \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<double>>, \
      ops::ActivationKernel<plat::CUDADeviceContext,                        \
                            ops::functor<plat::float16>>);                  \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type##_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,   \
                                                 ops::grad_functor<float>>, \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<double>>,                 \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<plat::float16>>);

FOR_EACH_ACTIVATION_OP(REGISTER_ACTIVATION_CUDA_KERNEL);

/* ======================== leaky relu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(leaky_relu, LeakyRelu, LeakyReluFunctor,
                                LeakyReluGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    leaky_relu_grad_grad,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<
        plat::CUDADeviceContext, ops::LeakyReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ======================== elu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(elu, ELU, ELUFunctor, ELUGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    elu_grad_grad, ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                                            ops::ELUGradGradFunctor<float>>,
    ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                             ops::ELUGradGradFunctor<double>>,
    ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                             ops::ELUGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================    relu register  ============================ */
REGISTER_OP_CUDA_KERNEL(
    relu, ops::ReluBaseKernel<paddle::platform::CUDADeviceContext,
                              ops::ReluGPUFunctor<float, 2>>,
    ops::ReluBaseKernel<paddle::platform::CUDADeviceContext,
                        ops::ReluGPUFunctor<double, 4>>,
    ops::ReluBaseKernel<paddle::platform::CUDADeviceContext,
                        ops::ReluGPUFunctor<plat::float16, 1>>);

REGISTER_OP_CUDA_KERNEL(
    relu_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                         ops::ReluGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ReluGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ReluGradFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    relu_grad_grad,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   sqrt register  ============================= */
REGISTER_ACTIVATION_CUDA_KERNEL(sqrt, Sqrt, SqrtFunctor, SqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    sqrt_grad_grad,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<float>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<double>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   rsqrt register  =============================
 */
REGISTER_ACTIVATION_CUDA_KERNEL(rsqrt, Rsqrt, RsqrtFunctor, RsqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    rsqrt_grad_grad,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<float>>,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<double>>,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================  square register  ============================ */
REGISTER_OP_CUDA_KERNEL(
    square,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<float>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<double>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<int>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<int64_t>>,
    ops::ActivationKernel<plat::CUDADeviceContext,
                          ops::SquareFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    square_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                           ops::SquareGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<int>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<int64_t>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<plat::float16>>);

REGISTER_OP_CUDA_KERNEL(
    square_grad_grad,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<float>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<double>>,
    ops::SquareDoubleGradKernel<plat::CUDADeviceContext,
                                ops::SquareGradGradFunctor<plat::float16>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<int>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<int64_t>>);
/* ========================================================================== */

/* ==========================   pow register  ============================ */

REGISTER_OP_CUDA_KERNEL(
    pow, ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<float>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<double>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<int>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<int64_t>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    pow_grad,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<float>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<double>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<int>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<int64_t>>,
    ops::PowGradKernel<plat::CUDADeviceContext,
                       ops::PowGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ==========================   exp register  ============================ */

REGISTER_OP_CUDA_KERNEL(
    exp, ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<float>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<double>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<int>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<int64_t>>,
    ops::ActivationKernel<plat::CUDADeviceContext,
                          ops::ExpFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    exp_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                        ops::ExpGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<int>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<int64_t>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ==========================  Log register ==================================*/
REGISTER_ACTIVATION_CUDA_KERNEL(log, Log, LogFunctor, LogGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    log_grad_grad, ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                                            ops::LogGradGradFunctor<float>>,
    ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                             ops::LogGradGradFunctor<double>>,
    ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                             ops::LogGradGradFunctor<plat::float16>>);
/* ========================================================================== */
