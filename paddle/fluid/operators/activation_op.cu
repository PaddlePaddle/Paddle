#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using float16 = paddle::platform::float16;

template <typename T>
struct CudaVecType {
  using type = T;
  static constexpr int vecsize = 1;
};

template <>
struct CudaVecType<platform::float16> {
  using type = __half2;
  static constexpr int vecsize = 2;
};

template <>
struct CudaVecType<float> {
  using type = float4;
  static constexpr int vecsize = 4;
};

template <typename T>
class BaseGPUFunctor {
 public:
  using ELEMENT_TYPE = T;
};

/* ========================================================================== */

/* ===========================    relu forward   ============================ */
template <typename T>
class ReluGPUFuctor : public BaseGPUFunctor<T> {
 private:
  T zero_;

 public:
  ReluGPUFuctor() { zero_ = static_cast<T>(0.0f); }

  // for relu forward when T is double
  __device__ __forceinline__ typename CudaVecType<T>::type Compute(
      const typename CudaVecType<T>::type* x);

  // when num % vecsize != 0 this func will be used
  __device__ __forceinline__ T ComputeRemainder(const T x) {
    return x > zero_ ? x : zero_;
  }
};

template <>
__device__ __forceinline__ CudaVecType<double>::type
ReluGPUFuctor<double>::Compute(const CudaVecType<double>::type* x) {
// relu forward : out = max(x, 0)
#ifdef __HIPCC__ || __CUDA_ARCH__ >= 350 || CUDA_VERSION >= 300
  return __ldg(x) > zero_ ? __ldg(x) : zero_;
#else
  return (*x) > zero_ ? (*x) : zero_;
#endif
}

template <>
__device__ __forceinline__ CudaVecType<float>::type
ReluGPUFuctor<float>::Compute(const CudaVecType<float>::type* xx) {
  // relu forward : out = max(xx, 0)
  return make_float4((xx->x > zero_) * (xx->x), (xx->y > zero_) * (xx->y),
                     (xx->z > zero_) * (xx->z), (xx->w > zero_) * (xx->w));
}

template <>
__device__ __forceinline__ CudaVecType<float16>::type
ReluGPUFuctor<float16>::Compute(const CudaVecType<float16>::type* in) {
// relu forward : out = max(in, 0)
#ifdef __HIPCC__ || __CUDA_ARCH__ >= 350 || CUDA_VERSION >= 300
  const half2 kzero = __float2half2_rn(0.0f);
  return __hmul2(__hgt2(__ldg(in), kzero), __ldg(in));
#else
  const float2 xx = __half22float2(*in);
  return __floats2half2_rn((xx.x > 0.0f) * static_cast<float>(xx.x),
                           (xx.y > 0.0f) * static_cast<float>(xx.y));
#endif
}
/* ========================================================================== */

/* ===========================    relu backward   ============================
 */

template <typename T>
class ReluGradGPUFunctor : public BaseGPUFunctor<T> {
 private:
  T zero_;

 public:
  ReluGradGPUFunctor() { zero_ = static_cast<T>(0.0f); }

  // for relu backward when T is double
  __device__ __forceinline__ typename CudaVecType<T>::type Compute(
      const typename CudaVecType<T>::type* out,
      const typename CudaVecType<T>::type* dout);

  // when num % vecsize != 0 this func will be used
  __device__ __forceinline__ T ComputeRemainder(const T out, const T dout) {
    // relu backward : dx = out > 0 ? dout : 0;
    return out > zero_ ? dout : zero_;
  }

  static constexpr ActBwdOpFwdDeps FwdDeps() { return kDepOut; }
};

template <>
__device__ __forceinline__ CudaVecType<double>::type
ReluGradGPUFunctor<double>::Compute(const CudaVecType<double>::type* out,
                                    const CudaVecType<double>::type* dout) {
// relu backward : dx = out > 0 ? dout : 0;
#ifdef __HIPCC__ || __CUDA_ARCH__ >= 350 || CUDA_VERSION >= 300
  return __ldg(out) > zero_ ? __ldg(dout) : zero_;
#else
  return (*out) > zero_ ? (*dout) : zero_;
#endif
}

template <>
__device__ __forceinline__ CudaVecType<float>::type
ReluGradGPUFunctor<float>::Compute(const CudaVecType<float>::type* out,
                                   const CudaVecType<float>::type* dout) {
  // relu backward : dx = out > 0 ? dout : 0;
  return make_float4((out->x > zero_) * (dout->x), (out->y > zero_) * (dout->y),
                     (out->z > zero_) * (dout->z),
                     (out->w > zero_) * (dout->w));
}

template <>
__device__ __forceinline__ CudaVecType<float16>::type
ReluGradGPUFunctor<float16>::Compute(const CudaVecType<float16>::type* out,
                                     const CudaVecType<float16>::type* dout) {
// relu backward : dx = out > 0 ? dout : 0;
#ifdef __HIPCC__ || __CUDA_ARCH__ >= 350 || CUDA_VERSION >= 300
  const half2 kzero = __float2half2_rn(0.0f);
  return __hmul2(__hgt2(__ldg(out), kzero), __ldg(dout));
#else
  const float2 xx = __half22float2(*out);
  const float2 yy = __half22float2(*dout);
  return __floats2half2_rn((xx.x > 0.0f) * static_cast<float>(yy.x),
                           (xx.y > 0.0f) * static_cast<float>(yy.y));
#endif
}

/* ========================================================================== */

template <typename T, typename Functor>
__global__ void ActivationGradKernelVec(const T* forward_data, const T* dout,
                                        T* dx, int num, Functor functor) {
  using VecType = typename CudaVecType<T>::type;
  constexpr int vecsize = CudaVecType<T>::vecsize;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  int loop = num / vecsize;
  int tail = num % vecsize;
  const VecType* in_forward = reinterpret_cast<const VecType*>(forward_data);
  const VecType* in_dout = reinterpret_cast<const VecType*>(dout);
  VecType* out = reinterpret_cast<VecType*>(dx);

  for (int i = idx; i < loop; i += stride) {
    out[i] = functor.Compute((in_forward + i), (in_dout + i));
  }

  while (idx == loop && tail) {
    dx[num - tail] =
        functor.ComputeRemainder(forward_data[num - tail], dout[num - tail]);
    --tail;
  }
}

template <typename T, typename Functor>
__global__ void ActivationkernelVec(const T* src, T* dst, int num,
                                    Functor functor) {
  constexpr int vecsize = CudaVecType<T>::vecsize;
  using VecType = typename CudaVecType<T>::type;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  int loop = num / vecsize;
  int tail = num % vecsize;
  const VecType* in = reinterpret_cast<const VecType*>(src);
  VecType* out = reinterpret_cast<VecType*>(dst);

  for (int i = idx; i < loop; i += stride) {
    out[i] = functor.Compute((in + i));
  }

  while (idx == loop && tail) {
    dst[num - tail] = functor.ComputeRemainder(src[num - tail]);
    --tail;
  }
}

template <typename DeviceContext, typename Functor>
class ActivationGPUKernel
    : public framework::OpKernel<typename Functor::ELEMENT_TYPE> {
 public:
  using T = typename Functor::ELEMENT_TYPE;
  void Compute(const framework::ExecutionContext& context) const override {
    const framework::Tensor* in_x = nullptr;
    framework::Tensor* out = nullptr;
    ExtractActivationTensor(context, &in_x, &out);
    auto& dev_ctx = context.template device_context<DeviceContext>();

    int num = in_x->numel();
    const T* input_data = in_x->data<T>();
    T* output_data = out->mutable_data<T>(dev_ctx.GetPlace(),
                                          static_cast<size_t>(num * sizeof(T)));

    int block = 512;
#ifdef __HIPCC__
    block = 256;
#endif
    Functor functor;
    constexpr int vecsize = CudaVecType<T>::vecsize;
    int grid = max((num / vecsize + block - 1) / block, 1);
    ActivationkernelVec<T, Functor><<<grid, block>>>(input_data, output_data,
                                                     num, functor);
  }
};

template <typename DeviceContext, typename Functor>
class ActivationGradGPUKernel
    : public framework::OpKernel<typename Functor::ELEMENT_TYPE> {
 public:
  using T = typename Functor::ELEMENT_TYPE;
  void Compute(const framework::ExecutionContext& context) const override {
    const framework::Tensor *x, *out, *d_out;
    framework::Tensor* d_x = nullptr;
    x = out = d_out = nullptr;
    ExtractActivationGradTensor<Functor::FwdDeps()>(context, &x, &out, &d_out,
                                                    &d_x);
    int numel = d_out->numel();
    auto& dev_ctx = context.template device_context<DeviceContext>();
    auto* dx_data = d_x->mutable_data<T>(
        dev_ctx.GetPlace(), static_cast<size_t>(numel * sizeof(T)));
    auto* dout_data = d_out->data<T>();

    auto* forward_data = dout_data;
    if (static_cast<int>(Functor::FwdDeps()) == static_cast<int>(kDepOut)) {
      // Only need forward output Out
      forward_data = out->data<T>();
    } else if (static_cast<int>(Functor::FwdDeps()) ==
               static_cast<int>(kDepX)) {
      // Only need forward input X
      forward_data = x->data<T>();
    }

    int block = 512;
#ifdef __HIPCC__
    block = 256;
#endif
    Functor functor;
    constexpr int vecsize = CudaVecType<T>::vecsize;
    int grid = max((numel / vecsize + block - 1) / block, 1);
    ActivationGradKernelVec<T, Functor><<<grid, block>>>(
        forward_data, dout_data, dx_data, numel, functor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

#define REGISTER_ACTIVATION_CUDA_KERNEL(act_type, op_name, functor,         \
                                        grad_functor)                       \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type,                                                             \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<float>>,  \
      ops::ActivationKernel<plat::CUDADeviceContext, ops::functor<double>>, \
      ops::ActivationKernel<plat::CUDADeviceContext,                        \
                            ops::functor<plat::float16>>);                  \
  REGISTER_OP_CUDA_KERNEL(                                                  \
      act_type##_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,   \
                                                 ops::grad_functor<float>>, \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<double>>,                 \
      ops::ActivationGradKernel<plat::CUDADeviceContext,                    \
                                ops::grad_functor<plat::float16>>);

FOR_EACH_ACTIVATION_OP(REGISTER_ACTIVATION_CUDA_KERNEL);

/* ======================== leaky relu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(leaky_relu, LeakyRelu, LeakyReluFunctor,
                                LeakyReluGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    leaky_relu_grad_grad,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::LeakyReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<
        plat::CUDADeviceContext, ops::LeakyReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ======================== elu register  ============================ */
REGISTER_ACTIVATION_CUDA_KERNEL(elu, ELU, ELUFunctor, ELUGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    elu_grad_grad, ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                                            ops::ELUGradGradFunctor<float>>,
    ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                             ops::ELUGradGradFunctor<double>>,
    ops::ELUDoubleGradKernel<plat::CUDADeviceContext,
                             ops::ELUGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================    relu register  ============================ */
REGISTER_OP_CUDA_KERNEL(
    relu, ops::ActivationGPUKernel<paddle::platform::CUDADeviceContext,
                                   ops::ReluGPUFuctor<float>>,
    ops::ActivationGPUKernel<paddle::platform::CUDADeviceContext,
                             ops::ReluGPUFuctor<double>>,
    ops::ActivationGPUKernel<plat::CUDADeviceContext,
                             ops::ReluGPUFuctor<plat::float16>>);

REGISTER_OP_CUDA_KERNEL(
    relu_grad, ops::ActivationGradGPUKernel<paddle::platform::CUDADeviceContext,
                                            ops::ReluGradGPUFunctor<float>>,
    ops::ActivationGradGPUKernel<paddle::platform::CUDADeviceContext,
                                 ops::ReluGradGPUFunctor<double>>,
    ops::ActivationGradGPUKernel<plat::CUDADeviceContext,
                                 ops::ReluGradGPUFunctor<plat::float16>>);

REGISTER_OP_CUDA_KERNEL(
    relu_grad_grad,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<float>>,
    ops::ActivationDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<double>>,
    ops::ActivationDoubleGradKernel<plat::CUDADeviceContext,
                                    ops::ReluGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   sqrt register  ============================= */
REGISTER_ACTIVATION_CUDA_KERNEL(sqrt, Sqrt, SqrtFunctor, SqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    sqrt_grad_grad,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<float>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<double>>,
    ops::SqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                              ops::SqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================   rsqrt register  =============================
 */
REGISTER_ACTIVATION_CUDA_KERNEL(rsqrt, Rsqrt, RsqrtFunctor, RsqrtGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    rsqrt_grad_grad,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<float>>,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<double>>,
    ops::RsqrtDoubleGradKernel<paddle::platform::CUDADeviceContext,
                               ops::RsqrtGradGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ===========================  square register  ============================ */
REGISTER_OP_CUDA_KERNEL(
    square,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<float>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<double>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<int>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::SquareFunctor<int64_t>>,
    ops::ActivationKernel<plat::CUDADeviceContext,
                          ops::SquareFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    square_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                           ops::SquareGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<int>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<int64_t>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::SquareGradFunctor<plat::float16>>);

REGISTER_OP_CUDA_KERNEL(
    square_grad_grad,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<float>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<double>>,
    ops::SquareDoubleGradKernel<plat::CUDADeviceContext,
                                ops::SquareGradGradFunctor<plat::float16>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<int>>,
    ops::SquareDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                ops::SquareGradGradFunctor<int64_t>>);
/* ========================================================================== */

/* ==========================   pow register  ============================ */

REGISTER_OP_CUDA_KERNEL(
    pow, ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<float>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<double>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<int>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<int64_t>>,
    ops::PowKernel<plat::CUDADeviceContext, ops::PowFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    pow_grad,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<float>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<double>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<int>>,
    ops::PowGradKernel<plat::CUDADeviceContext, ops::PowGradFunctor<int64_t>>,
    ops::PowGradKernel<plat::CUDADeviceContext,
                       ops::PowGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ==========================   exp register  ============================ */

REGISTER_OP_CUDA_KERNEL(
    exp, ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<float>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<double>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<int>>,
    ops::ActivationKernel<plat::CUDADeviceContext, ops::ExpFunctor<int64_t>>,
    ops::ActivationKernel<plat::CUDADeviceContext,
                          ops::ExpFunctor<plat::float16>>);
REGISTER_OP_CUDA_KERNEL(
    exp_grad, ops::ActivationGradKernel<plat::CUDADeviceContext,
                                        ops::ExpGradFunctor<float>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<double>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<int>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<int64_t>>,
    ops::ActivationGradKernel<plat::CUDADeviceContext,
                              ops::ExpGradFunctor<plat::float16>>);
/* ========================================================================== */

/* ==========================  Log register ==================================*/
REGISTER_ACTIVATION_CUDA_KERNEL(log, Log, LogFunctor, LogGradFunctor);

REGISTER_OP_CUDA_KERNEL(
    log_grad_grad, ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                                            ops::LogGradGradFunctor<float>>,
    ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                             ops::LogGradGradFunctor<double>>,
    ops::LogDoubleGradKernel<plat::CUDADeviceContext,
                             ops::LogGradGradFunctor<plat::float16>>);
/* ========================================================================== */
