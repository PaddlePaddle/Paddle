#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include "paddle/fluid/operators/quad_transform_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;

template <int T>
__global__ void QuadTransformKernel(const int n, const int h, const int w,
                                    const T* input, T* output) {
  int id_n = threadIdx.x + blockDim.x * blockIdx.x;
  int id_h = threadIdx.y + blockDim.y * blockIdx.y;
  int id_w = threadIdx.z + blockDim.z * blockIdx.z;
  if (idx < n && idy < h && idz < w) {
    int id = id_n * h * w + w * id_h + id_w;
    if (id_n % 2 == 0) {
      output[id] = input[id] + id_w;
    } else {
      output[id] = input[id] + id_h;
    }
  }
}

template <typename T>
class QuadTransfromOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto* in = ctx.Input<Tensor>("Input");
    auto in_dims = in->dims();
    const T* in_data = in->data<T>();
    auto* out = ctx.Output<Tensor>("Output");
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    int batch_size = in_dims[0];
    int height = in_dims[2];
    int width = in_dims[3];
    dim3 threadsPerBlock(4, 16, 16);
    dim3 numBlocks((batch_size * 8) / threadsPerBlock.x,
                   height / threadsPerBlock.y, width / threadsPerBlock.z);
    QuadTransfromCudaKernel<T><<<numBlocks, threadsPerBlock, 0, stream>>>(
        batch_size * 8, height, width, in_data, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(quad_transform, paddle::operators::OpCUDAKernel<float>,
                        paddle::operators::AccuracyOpCUDAKernel<double>);
