#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/index_select_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T, typename IndexT>
__global__ void index_select_cuda_kernel(const T* input, T* output,
                                         const IndexT* index, int64_t N,
                                         int64_t stride, int64_t size,
                                         int64_t delta) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t pre_idx = idx / (stride * size);
  int64_t dim_idx = idx % (stride * size) / stride;
  IndexT src_dim_idx = index[dim_idx];
  int64_t input_idx = idx + (delta * pre_idx + src_dim_idx - dim_idx) * stride;
  output[idx] = input[input_idx];
}

template <typename T, typename IndexT>
__global__ void index_select_grad_cuda_kernel(const T* output_grad,
                                              T* input_grad,
                                              const IndexT* index, int64_t nums,
                                              int64_t N, int64_t stride,
                                              int64_t size, int64_t delta) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t pre_idx = idx / (stride * size);
  int64_t dim_idx = idx % (stride * size) / stride;
  IndexT src_dim_idx = index[dim_idx];
  int64_t input_idx = idx + (delta * pre_idx + src_dim_idx - dim_idx) * stride;
  paddle::platform::CudaAtomicAdd(&input_grad[input_idx], output_grad[idx]);
}

template <typename T>
__global__ void index_select_grad_init(T* input_grad, int64_t N) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }
  input_grad[idx] = 0.0;
}

template <typename DeviceContext, typename T>
class IndexSelectCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* index = context.Input<LoDTensor>("Index");
    auto* out = context.Output<LoDTensor>("Out");
    int dim = context.Attr<int>("dim");
    auto input_dim = in->dims();
    auto output_dim = out->dims();
    dim = dim >= 0 ? dim : dim + input_dim.size();
    auto stride_dim = phi::stride(input_dim);
    int64_t stride = stride_dim[dim];
    int64_t size = output_dim[dim];
    int64_t delta = input_dim[dim] - size;

    const auto& index_type = framework::TransToProtoVarType(index->dtype());
    bool index_type_match = index_type == framework::proto::VarType::INT64 ||
                            index_type == framework::proto::VarType::INT32;
    PADDLE_ENFORCE_EQ(index_type_match, true,
                      platform::errors::InvalidArgument(
                          "Input(Index) holds the wrong type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(index_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = out->numel();

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    if (index_type == framework::proto::VarType::INT64) {
      const int64_t* index_data = index->data<int64_t>();
      index_select_cuda_kernel<T, int64_t><<<
          (numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_data, out_data, index_data,
                                                numel, stride, size, delta);
      platform::GpuStreamSync(stream);
    } else {
      const int* index_data = index->data<int>();
      index_select_cuda_kernel<T, int><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                                             PADDLE_CUDA_NUM_THREADS,
                                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          in_data, out_data, index_data, numel, stride, size, delta);
      platform::GpuStreamSync(stream);
    }
  }
};

template <typename DeviceContext, typename T>
class IndexSelectGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* output_grad = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* in_grad = context.Output<LoDTensor>(framework::GradVarName("X"));
    auto* index = context.Input<LoDTensor>("Index");

    auto* output_grad_data = output_grad->data<T>();
    auto* in_grad_data = in_grad->mutable_data<T>(context.GetPlace());

    int dim = context.Attr<int>("dim");
    auto input_dim = in_grad->dims();
    auto output_dim = output_grad->dims();
    dim = dim >= 0 ? dim : dim + input_dim.size();
    auto stride_dim = phi::stride(input_dim);
    int64_t stride = stride_dim[dim];
    int64_t size = output_dim[dim];
    int64_t delta = input_dim[dim] - size;

    const auto& index_type = framework::TransToProtoVarType(index->dtype());
    bool index_type_match = index_type == framework::proto::VarType::INT64 ||
                            index_type == framework::proto::VarType::INT32;
    PADDLE_ENFORCE_EQ(index_type_match, true,
                      platform::errors::InvalidArgument(
                          "Input(Index) holds the wrong type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(index_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    int64_t numel = in_grad->numel();
    int64_t index_nums = index->numel();
    int64_t out_nums = output_grad->numel();

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    index_select_grad_init<
        T><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_grad_data, numel);

    if (index_type == framework::proto::VarType::INT64) {
      const int64_t* index_data = index->data<int64_t>();
      index_select_grad_cuda_kernel<T, int64_t><<<
          (out_nums + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(output_grad_data, in_grad_data,
                                                index_data, index_nums,
                                                out_nums, stride, size, delta);
      platform::GpuStreamSync(stream);
    } else {
      const int* index_data = index->data<int>();
      index_select_grad_cuda_kernel<T, int><<<
          (out_nums + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(output_grad_data, in_grad_data,
                                                index_data, index_nums,
                                                out_nums, stride, size, delta);
      platform::GpuStreamSync(stream);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    index_select,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext,
                               paddle::platform::float16>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    index_select_grad,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                   paddle::platform::float16>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                   int64_t>);
