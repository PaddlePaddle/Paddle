#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/index_select_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T, typename IndexT>
__global__ void index_select_cuda_kernel(const T* input, T* output,
                                         const IndexT* index, int64_t N,
                                         int64_t stride, int64_t size,
                                         int64_t delta) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t pre_idx = idx / (stride * size);
  int64_t dim_idx = idx % (stride * size) / stride;
  IndexT src_dim_idx = index[dim_idx];
  int64_t input_idx = idx + (delta * pre_idx + src_dim_idx - dim_idx) * stride;
  output[idx] = input[input_idx];
}

template <typename T, typename IndexT>
__global__ void index_select_grad_cuda_kernel(const T* output_grad,
                                              T* input_grad,
                                              const IndexT* index, int64_t nums,
                                              int64_t N, int64_t stride,
                                              int64_t size, int64_t delta) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t pre_idx = idx / (stride * size);
  int64_t dim_idx = idx % (stride * size) / stride;
  int64_t begin_idx = idx + (delta * pre_idx - dim_idx) * stride;

  input_grad[idx] = 0.0;
  for (int64_t i = 0; i < nums; i++) {
    if (index[i] == dim_idx) {
      input_grad[idx] += output_grad[begin_idx + i * stride];
    }
  }
}

template <typename DeviceContext, typename T>
class IndexSelectCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* index = context.Input<LoDTensor>("Index");
    auto* out = context.Output<LoDTensor>("Out");
    int dim = context.Attr<int>("dim");
    auto input_dim = in->dims();
    auto output_dim = out->dims();
    dim = dim >= 0 ? dim : dim + input_dim.size();
    auto stride_dim = framework::stride(input_dim);
    int64_t stride = stride_dim[dim];
    int64_t size = output_dim[dim];
    int64_t delta = input_dim[dim] - size;

    const auto& index_type = index->type();
    bool index_type_match = index_type == framework::proto::VarType::INT64 ||
                            index_type == framework::proto::VarType::INT32;
    PADDLE_ENFORCE_EQ(index_type_match, true,
                      platform::errors::InvalidArgument(
                          "Input(Index) holds the wrong type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(index_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = out->numel();

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    if (index_type == framework::proto::VarType::INT64) {
      const int64_t* index_data = index->data<int64_t>();
      index_select_cuda_kernel<T, int64_t><<<
          (numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_data, out_data, index_data,
                                                numel, stride, size, delta);
#ifdef PADDLE_WITH_HIP
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#else
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#endif
    } else {
      const int* index_data = index->data<int>();
      index_select_cuda_kernel<T, int><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                                             PADDLE_CUDA_NUM_THREADS,
                                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          in_data, out_data, index_data, numel, stride, size, delta);
#ifdef PADDLE_WITH_HIP
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#else
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#endif
    }
  }
};

template <typename DeviceContext, typename T>
class IndexSelectGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* output_grad = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* in_grad = context.Output<LoDTensor>(framework::GradVarName("X"));
    auto* index = context.Input<LoDTensor>("Index");

    auto* output_grad_data = output_grad->data<T>();
    auto* in_grad_data = in_grad->mutable_data<T>(context.GetPlace());

    int dim = context.Attr<int>("dim");
    auto input_dim = in_grad->dims();
    auto output_dim = output_grad->dims();
    dim = dim >= 0 ? dim : dim + input_dim.size();
    auto stride_dim = framework::stride(input_dim);
    int64_t stride = stride_dim[dim];
    int64_t size = input_dim[dim];
    int64_t delta = output_dim[dim] - size;

    const auto& index_type = index->type();
    bool index_type_match = index_type == framework::proto::VarType::INT64 ||
                            index_type == framework::proto::VarType::INT32;
    PADDLE_ENFORCE_EQ(index_type_match, true,
                      platform::errors::InvalidArgument(
                          "Input(Index) holds the wrong type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(index_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    int64_t numel = in_grad->numel();
    int64_t index_nums = index->numel();

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    if (index_type == framework::proto::VarType::INT64) {
      const int64_t* index_data = index->data<int64_t>();
      index_select_grad_cuda_kernel<T, int64_t><<<
          (numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(output_grad_data, in_grad_data,
                                                index_data, index_nums, numel,
                                                stride, size, delta);
#ifdef PADDLE_WITH_HIP
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#else
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#endif
    } else {
      const int* index_data = index->data<int>();
      index_select_grad_cuda_kernel<T, int><<<
          (numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(output_grad_data, in_grad_data,
                                                index_data, index_nums, numel,
                                                stride, size, delta);
#ifdef PADDLE_WITH_HIP
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#else
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
#endif
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    index_select,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    index_select_grad,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                   int64_t>);
