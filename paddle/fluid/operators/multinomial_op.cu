#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifndef PADDLE_WITH_HIP
// To-do(qili93): fix this after issue resolved
// https://github.com/ROCmSoftwarePlatform/rocPRIM/issues/202

#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/multinomial_op.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/transform.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void NormalizeProbability(T* norm_probs, const T* in_data,
                                     T* sum_rows) {
  int id = threadIdx.x + blockIdx.x * blockDim.x +
           blockIdx.y * gridDim.x * blockDim.x;
  PADDLE_ENFORCE(
      in_data[id] >= 0.0,
      "The input of multinomial distribution should be >= 0, but got %f.",
      in_data[id]);
  PADDLE_ENFORCE(sum_rows[blockIdx.y] > 0.0,
                 "The sum of one multinomial distribution probability should "
                 "be > 0, but got %f.",
                 sum_rows[blockIdx.y]);
  norm_probs[id] = in_data[id] / sum_rows[blockIdx.y];
}

template <typename T>
__global__ void GetCumulativeProbs(T* norm_probs_data,
                                   int64_t num_distributions,
                                   int64_t num_categories,
                                   T* cumulative_probs) {
  for (int id = blockIdx.x; id < num_distributions; id += gridDim.x) {
    thrust::inclusive_scan(thrust::device,
                           norm_probs_data + id * num_categories,
                           norm_probs_data + (id + 1) * num_categories,
                           cumulative_probs + id * num_categories);
  }
}

template <typename T>
struct RandomGeneratorCudaFunctor {
  unsigned int seed_;
  __host__ __device__ RandomGeneratorCudaFunctor(int seed) : seed_(seed) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(0.0, 1.0);
    rng.discard(n);
    return dist(rng);
  }
};

template <typename T>
__device__ int binarySearchFunctor(T* cumulative_probs, T* norm_probs_data,
                                   int num_categories, T rng_number) {
  int left = 0;
  int right = num_categories;

  while (right - left > 0) {
    int mid = left + (right - left) / 2;

    T temp_prob = cumulative_probs[mid];
    if (temp_prob < rng_number) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }

  if (left == num_categories) {
    left = num_categories - 1;
  }

  while (left >= 1 && norm_probs_data[left] == 0) left--;

  return left;
}

template <typename T>
__global__ void sampleMultinomialWithReplacement(
    T* rng_data, const int64_t num_samples, int64_t* out_data,
    const int64_t num_distributions, const int64_t num_categories,
    T* cumulative_probs, T* norm_probs_data) {
  // use binary search to get the selected category sample id.
  // let cumulative_probs[id-1] < rng_data < cumulative_probs[id].

  int idx = threadIdx.x + blockIdx.x * blockDim.x +
            blockIdx.y * gridDim.x * blockDim.x;

  // for every distribution
  for (int dist = blockIdx.y; dist < num_distributions; dist += gridDim.y) {
    // for every sample
    for (int sample = blockIdx.x * blockDim.x + threadIdx.x;
         sample < num_samples; sample += blockDim.x * gridDim.x) {
      T rng_number = rng_data[sample + dist * num_samples];

      // Find the bucket that a uniform random number lies in
      int selected_category = binarySearchFunctor<T>(
          cumulative_probs + dist * num_categories,
          norm_probs_data + dist * num_categories, num_categories, rng_number);

      out_data[sample + dist * num_samples] = selected_category;
    }
  }
}

template <typename T>
class MultinomialOpKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const auto x = ctx.Input<framework::Tensor>("X");
    auto out = ctx.Output<framework::Tensor>("Out");

    const int64_t num_samples = ctx.Attr<int>("num_samples");
    const bool replacement = ctx.Attr<bool>("replacement");

    auto* in_data = x->data<T>();
    int64_t* out_data = out->mutable_data<int64_t>(ctx.GetPlace());

    auto in_dims = x->dims();
    int64_t in_rank = in_dims.size();
    const int64_t num_categories = in_dims[in_rank - 1];
    const int64_t num_distributions = in_rank > 1 ? in_dims[in_rank - 2] : 1;

    // If replacement is False, it's not a replaceable sample. Every category
    // can
    // be used only once. So after every sample, probability of the distribution
    // will change. The implementation can't be parallelizable. Thus, call CPU
    // implementation ``MultinomialFunctor`` to sample the distribution.
    if (!replacement) {
      int64_t in_data_numel = x->numel();
      int64_t out_data_numel = out->numel();

      T* cpu_in_data = new T[in_data_numel];
      int64_t* cpu_out_data = new int64_t[out_data_numel];

#ifdef PADDLE_WITH_HIP
      hipMemcpy(cpu_in_data, in_data, in_data_numel * sizeof(T),
                hipMemcpyDeviceToHost);
#else
      hipMemcpy(cpu_in_data, in_data, in_data_numel * sizeof(T),
                 hipMemcpyDeviceToHost);
#endif

      MultinomialFunctor<T>(cpu_out_data, cpu_in_data, num_samples, replacement,
                            num_categories, num_distributions);

#ifdef PADDLE_WITH_HIP
      hipMemcpy(out_data, cpu_out_data, out_data_numel * sizeof(int64_t),
                hipMemcpyHostToDevice);
#else
      hipMemcpy(out_data, cpu_out_data, out_data_numel * sizeof(int64_t),
                 hipMemcpyHostToDevice);
#endif

      delete[] cpu_in_data;
      delete[] cpu_out_data;
      return;
    }

    // Sum of input may not be 1. To get probability in range [0, 1], calculate
    // sum of each row of input, and then use the sum to normalize the input.
    // sum_row_data: sum of each row
    framework::Tensor sum_rows_tensor;
    auto* sum_rows_data =
        sum_rows_tensor.mutable_data<T>({num_distributions}, ctx.GetPlace());

    auto& place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();

    if (num_distributions == 1) {
      auto eigen_input = framework::EigenVector<T>::Flatten(*x);
      auto eigen_sum_rows = framework::EigenVector<T>::Flatten(sum_rows_tensor);
      eigen_sum_rows.device(place) =
          eigen_input.sum(Eigen::DSizes<int, 1>(1))
              .eval()
              .reshape(Eigen::DSizes<int, 1>(sum_rows_tensor.dims()[0]));
    } else {
      auto eigen_input = framework::EigenMatrix<T>::From(*x);
      auto eigen_sum_rows = framework::EigenVector<T>::Flatten(sum_rows_tensor);
      eigen_sum_rows.device(place) = eigen_input.sum(Eigen::DSizes<int, 1>(1));
    }

    // Normalize row of each distribution to get the probability in range [0,
    // 1].
    // norm_probs_data: probability of the distribution
    framework::Tensor norm_probs_tensor;
    auto* norm_probs_data = norm_probs_tensor.mutable_data<T>(
        {num_distributions, num_categories}, ctx.GetPlace());

    // number of threads in a block is min(num_categories, 512)
    dim3 block_norm(num_categories < 512 ? num_categories : 512);
    dim3 grid_norm((num_categories - 1) / block_norm.x + 1, num_distributions);
    NormalizeProbability<
        T><<<grid_norm, block_norm, 0, ctx.cuda_device_context().stream()>>>(
        norm_probs_data, in_data, sum_rows_data);

    // Get cumulative probability of each distribution. It's the same function
    // of
    // ``cumsum`` op.
    framework::Tensor cumulative_probs_tensor;
    auto* cumulative_probs = cumulative_probs_tensor.mutable_data<T>(
        {num_distributions, num_categories}, ctx.GetPlace());
    dim3 block_cumsum(1);
    dim3 grid_cumsum(num_distributions);
    GetCumulativeProbs<T><<<grid_cumsum, block_cumsum, 0,
                            ctx.cuda_device_context().stream()>>>(
        norm_probs_data, num_distributions, num_categories, cumulative_probs);

    // Generate random number for each sample.
    std::random_device rd;
    auto seed = rd();

    framework::Tensor rng_data_tensor;
    auto* rng_data = rng_data_tensor.mutable_data<T>(
        {num_distributions, num_samples}, ctx.GetPlace());

    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    platform::Transform<platform::CUDADeviceContext> trans;
    auto* context =
        static_cast<const platform::CUDADeviceContext*>(&ctx.device_context());
    trans(*context, index_sequence_begin,
          index_sequence_begin + num_distributions * num_samples, rng_data,
          RandomGeneratorCudaFunctor<T>(seed));

    // Sample the multinomial distributions.
    dim3 block_sample(128);
    dim3 grid_sample((num_samples - 1) / block_sample.x + 1, num_distributions);
    sampleMultinomialWithReplacement<T><<<grid_sample, block_sample, 0,
                                          ctx.cuda_device_context().stream()>>>(
        rng_data, num_samples, out_data, num_distributions, num_categories,
        cumulative_probs, norm_probs_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    multinomial, ops::MultinomialOpKernel<plat::CUDADeviceContext, double>,
    ops::MultinomialOpKernel<plat::CUDADeviceContext, float>);

#endif
