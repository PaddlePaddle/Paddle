#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/norm_utils.cu.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/pten/kernels/funcs/math_function.h"

DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;



template <typename T>
class BatchNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    
  }
};



template <typename T>
class BatchNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {

    PADDLE_ENFORCE_EQ(
        x_dims.size() >= 2 && x_dims.size() <= 5, true,
        platform::errors::InvalidArgument(
            "The size of input's dimensions should be between 2 and 5."
            "But received: the size of input's dimensions is [%d],"
            "the dimensions of input is [%s]",
            x_dims.size(), x_dims));
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    // init output
    if (d_x) {
      d_x->mutable_data<T>(ctx.GetPlace());
    }

    if (d_scale && d_bias) {
      d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(
        scale->dims().size(), 1UL,
        platform::errors::InvalidArgument(
            "The size of scale's dimensions must equal to 1. But received: "
            "the size of scale's dimensions is [%d], the dimensions of scale "
            "is [%s].",
            scale->dims().size(), scale->dims()));
    PADDLE_ENFORCE_EQ(
        scale->dims()[0], C,
        platform::errors::InvalidArgument(
            "The first dimension of scale must equal to Channels[%d]. But "
            "received: the first dimension of scale is [%d]",
            C, scale->dims()[0]));

    auto dtype = platform::CudnnDataType<T>::type;
    const auto *reserve_space = ctx.Input<Tensor>("ReserveSpace");
#ifdef PADDLE_WITH_HIP
    auto compute_format = data_layout == DataLayout::kNHWC ? DataLayout::kNHWC
                                                           : DataLayout::kNCHW;

// TODO(wangran16): wait for MIOpen to improve the performance of BN
// HIP do not support compute format of NHWC
// auto compute_format = DataLayout::kNCHW;
#else
    const bool fast_nhwc_batch_norm =
        dtype == HIPDNN_DATA_HALF && FLAGS_cudnn_batchnorm_spatial_persistent &&
        reserve_space != nullptr;
    auto compute_format =
        fast_nhwc_batch_norm && data_layout == DataLayout::kNHWC
            ? DataLayout::kNHWC
            : DataLayout::kNCHW;
#endif

    Tensor transformed_x(x->type());
    Tensor transformed_d_y(d_y->type());
    Tensor transformed_d_x;
    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW && x_dims.size() > 2) {
      VLOG(3) << "Transform input tensor from NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                           &transformed_x);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                          &transformed_x);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_y,
                                                           &transformed_d_y);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_y,
                                                          &transformed_d_y);
      if (d_x) {
        ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_x,
                                                             &transformed_d_x);
      }
    } else {
      transformed_x.ShareDataWith(*x);
      transformed_d_y.ShareDataWith(*d_y);
      if (d_x) {
        transformed_d_x.ShareDataWith(*d_x);
      }
    }

    std::vector<int> dims;
    std::vector<int> strides;
    if (compute_format == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * C * D, 1, W * D * C, D * C, C};
    }

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    const int num = transformed_x.numel();
#ifdef HIPCC
    const int block = 256;
#else
    const int block = 512;
#endif
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    int grid1 = (num + block - 1) / block;
    int grid2 = std::min(C, max_blocks);
    auto stream = dev_ctx.stream();
    InplaceHelper<T> inplace_functor;

    if (!use_global_stats) {
      if ((N * H * W * D) == 1) {
        if (d_x) {
          framework::TensorCopy(*d_y, ctx.GetPlace(), d_x);
        }
        pten::funcs::SetConstant<platform::CUDADeviceContext,
                                 BatchNormParamType<T>>
            functor;
        functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
        functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
        return;
      }

// ------------------- cudnn descriptors ---------------------
#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// miopenTensorDescriptor_t data_desc_;
// miopenTensorDescriptor_t bn_param_desc_;
// miopenBatchNormMode_t mode_;

// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenCreateTensorDescriptor(&data_desc_));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenCreateTensorDescriptor(&bn_param_desc_));
#else
      hipdnnTensorDescriptor_t data_desc_;
      hipdnnTensorDescriptor_t bn_param_desc_;
      hipdnnBatchNormMode_t mode_;

      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
#endif
      if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
        LOG(ERROR) << "Provided epsilon is smaller than "
                   << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                   << "HIPDNN_BN_MIN_EPSILON instead.";
      }
      epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// mode_ = miopenBNSpatial;
#elif CUDNN_VERSION_MIN(7, 0, 1)
      if (FLAGS_cudnn_batchnorm_spatial_persistent) {
        mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
      } else if (H == 1 && W == 1) {
        mode_ = HIPDNN_BATCHNORM_PER_ACTIVATION;
      } else {
        mode_ = HIPDNN_BATCHNORM_SPATIAL;
      }
#else
      if (H == 1 && W == 1) {
        mode_ = HIPDNN_BATCHNORM_PER_ACTIVATION;
      } else {
        mode_ = HIPDNN_BATCHNORM_SPATIAL;
      }
#endif  // CUDNN_VERSION_MIN(7, 0, 1)

#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::miopenSetTensorDescriptor(
//     data_desc_, CudnnDataType<T>::type,
//     x_dims.size() > 3 ? x_dims.size() : 4, const_cast<int *>(dims.data()),
//     const_cast<int *>(strides.data())));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDeriveBNTensorDescriptor(bn_param_desc_,
//                                                       data_desc_, mode_));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_, CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDeriveBNTensorDescriptor(bn_param_desc_,
                                                           data_desc_, mode_));
#endif

      const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
      const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
      const auto *saved_mean_data =
          saved_mean->template data<BatchNormParamType<T>>();
      const auto *saved_var_data =
          saved_var->template data<BatchNormParamType<T>>();

      if (is_inplace) {
        inplace_functor(compute_format, transformed_x.data<T>(),
                        scale->template data<BatchNormParamType<T>>(),
                        bias->template data<BatchNormParamType<T>>(),
                        saved_mean_data, saved_var_data, epsilon, C, H * W * D,
                        num, transformed_x.data<T>(), grid2, block, stream);
      }

      // This branch calls CUDNN APIs
      if (d_x && d_scale && d_bias) {
        bool called = false;
#if CUDNN_VERSION_MIN(7, 4, 1)
        called = true;
        size_t workspace_size = 0;
        void *workspace_ptr = nullptr;
        Tensor workspace_tensor;
        auto reserve_space_size = reserve_space->memory_size();
        // --------------- cudnn batchnorm workspace ---------------
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::
                cudnnGetBatchNormalizationBackwardExWorkspaceSize(
                    /*handle=*/dev_ctx.cudnn_handle(),
                    /*mode=*/mode_,
                    /*bnIps=*/CUDNN_BATCHNORM_OPS_BN,
                    /*xDesc=*/data_desc_,
                    /*yDesc=*/data_desc_,
                    /*dyDesc=*/data_desc_,
                    /*dzDesc=*/nullptr,
                    /*dxDesc=*/data_desc_,
                    /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                    /*activationDesc=*/nullptr,
                    /*sizeInBytes=*/&workspace_size));

        workspace_ptr = workspace_tensor.mutable_data(
            ctx.GetPlace(), transformed_x.type(), workspace_size);

        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::cudnnBatchNormalizationBackwardEx(
                /*handle=*/dev_ctx.cudnn_handle(),
                /*mode=*/mode_,
                /*bnOps=*/CUDNN_BATCHNORM_OPS_BN,
                /*alphaDataDiff=*/CudnnDataType<T>::kOne(),
                /*betaDataDiff=*/CudnnDataType<T>::kZero(),
                /*alphaParamDiff=*/CudnnDataType<T>::kOne(),
                /*betaParamDiff=*/CudnnDataType<T>::kZero(),
                /*xDesc=*/data_desc_,
                /*xData=*/transformed_x.template data<T>(),
                /*yDesc=*/nullptr,
                /*yData=*/nullptr,
                /*dyDesc=*/data_desc_,
                /*dyData=*/transformed_d_y.template data<T>(),
                /*dzDesc=*/nullptr,
                /*dzData=*/nullptr,
                /*dxDesc=*/data_desc_,
                /*dxData=*/transformed_d_x.template mutable_data<T>(
                    ctx.GetPlace()),
                /*dBnScaleBiasDesc=*/bn_param_desc_,
                /*bnScaleData=*/scale->template data<BatchNormParamType<T>>(),
                /*bnBiasData=*/nullptr,
                /*dBnScaleData=*/d_scale
                    ->template mutable_data<BatchNormParamType<T>>(
                        ctx.GetPlace()),
                /*dBnBiasData=*/d_bias
                    ->template mutable_data<BatchNormParamType<T>>(
                        ctx.GetPlace()),
                /*epsilon=*/epsilon,
                /*savedMean=*/saved_mean_data,
                /*savedInvVariance=*/saved_var_data,
                /*activationDesc=*/nullptr,
                /*workspace=*/workspace_ptr,
                /*workSpaceSizeInBytes=*/workspace_size,
                /*reserveSpace=*/const_cast<T *>(
                    reserve_space->template data<T>()),
                /*reserveSpaceSizeInBytes=*/reserve_space_size));
#endif  // CUDNN_VERSION_MIN(7, 4, 1)
        if (!called) {
#ifdef PADDLE_WITH_HIP
          if (compute_format == DataLayout::kNCHW) {
            BNBackward<
                T, block,
                DataLayout::kNCHW><<<grid2, block, 0, dev_ctx.stream()>>>(
                transformed_d_y.template data<T>(),
                transformed_x.template data<T>(),
                scale->template data<BatchNormParamType<T>>(), saved_mean_data,
                saved_var_data, C, N, H * W * D, epsilon,
                transformed_d_x.template data<T>(),
                d_scale->template mutable_data<BatchNormParamType<T>>(
                    ctx.GetPlace()),
                d_bias->template mutable_data<BatchNormParamType<T>>(
                    ctx.GetPlace()));
          } else {
            BNBackward<
                T, block,
                DataLayout::kNHWC><<<grid2, block, 0, dev_ctx.stream()>>>(
                transformed_d_y.template data<T>(),
                transformed_x.template data<T>(),
                scale->template data<BatchNormParamType<T>>(), saved_mean_data,
                saved_var_data, C, N, H * W * D, epsilon,
                transformed_d_x.template data<T>(),
                d_scale->template mutable_data<BatchNormParamType<T>>(
                    ctx.GetPlace()),
                d_bias->template mutable_data<BatchNormParamType<T>>(
                    ctx.GetPlace()));
          }

// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenBatchNormalizationBackward(
//         dev_ctx.cudnn_handle(), mode_, CudnnDataType<T>::kOne(),
//         CudnnDataType<T>::kZero(), CudnnDataType<T>::kOne(),
//         CudnnDataType<T>::kZero(), data_desc_,
//         transformed_x.template data<T>(), data_desc_,
//         transformed_d_y.template data<T>(), data_desc_,
//         transformed_d_x.template mutable_data<T>(ctx.GetPlace()),
//         bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
//         d_scale->template mutable_data<BatchNormParamType<T>>(
//             ctx.GetPlace()),
//         d_bias->template mutable_data<BatchNormParamType<T>>(
//             ctx.GetPlace()),
//         epsilon, saved_mean_data, saved_var_data));
#else
          PADDLE_ENFORCE_GPU_SUCCESS(
              platform::dynload::hipdnnBatchNormalizationBackward(
                  dev_ctx.cudnn_handle(), mode_, CudnnDataType<T>::kOne(),
                  CudnnDataType<T>::kZero(), CudnnDataType<T>::kOne(),
                  CudnnDataType<T>::kZero(), data_desc_,
                  transformed_x.template data<T>(), data_desc_,
                  transformed_d_y.template data<T>(), data_desc_,
                  transformed_d_x.template mutable_data<T>(ctx.GetPlace()),
                  bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
                  d_scale->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  d_bias->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  epsilon, saved_mean_data, saved_var_data));
#endif
        }

        if (data_layout == DataLayout::kNHWC &&
            compute_format == DataLayout::kNCHW) {
          VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
          TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
              ctx, &transformed_d_x, d_x);
        }
      } else {
        // This branch call CUDA kernels
        if (compute_format == DataLayout::kNCHW) {
          if (d_x) {
            BNBackwardData<T, block, framework::DataLayout::kNCHW><<<
                grid2, block, 0, dev_ctx.stream()>>>(
                d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
                saved_mean_data, x->data<T>(), saved_var_data, C, N, H * W * D,
                d_x->data<T>());
          }
          if (d_scale && d_bias) {
            KeBNBackwardScaleBias<
                T, block,
                framework::DataLayout::kNCHW><<<grid2, block, 0, stream>>>(
                d_y->data<T>(), x->data<T>(), saved_mean_data, saved_var_data,
                epsilon, N, C, H * W * D,
                d_scale->data<BatchNormParamType<T>>(),
                d_bias->data<BatchNormParamType<T>>());
          }
        } else {
          if (d_x) {
            BNBackwardData<T, block, framework::DataLayout::kNHWC><<<
                grid2, block, 0, dev_ctx.stream()>>>(
                d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
                saved_mean_data, x->data<T>(), saved_var_data, C, N, H * W * D,
                d_x->data<T>());
          }
          if (d_scale && d_bias) {
            KeBNBackwardScaleBias<
                T, block,
                framework::DataLayout::kNHWC><<<grid2, block, 0, stream>>>(
                d_y->data<T>(), x->data<T>(), saved_mean_data, saved_var_data,
                epsilon, N, C, H * W * D,
                d_scale->data<BatchNormParamType<T>>(),
                d_bias->data<BatchNormParamType<T>>());
          }
        }
      }

#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// clean when exit.
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDestroyTensorDescriptor(data_desc_));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDestroyTensorDescriptor(bn_param_desc_));
#else
      // clean when exit.
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
#endif
    } else {
      const auto *running_mean = ctx.Input<Tensor>("Mean");
      const auto *running_var = ctx.Input<Tensor>("Variance");

      const auto *running_mean_data =
          running_mean->template data<BatchNormParamType<T>>();
      const auto *running_var_data =
          running_var->template data<BatchNormParamType<T>>();

      if (is_inplace) {
        auto px = *x;
        inplace_functor(data_layout, px.mutable_data<T>(ctx.GetPlace()),
                        scale->template data<BatchNormParamType<T>>(),
                        bias->template data<BatchNormParamType<T>>(),
                        running_mean_data, running_var_data, epsilon, C,
                        H * W * D, num, x->data<T>(), grid2, block, stream);
      }

      if (compute_format == DataLayout::kNCHW) {
        if (d_x) {
          KeBNBackwardData<
              T, framework::DataLayout::kNCHW><<<grid1, block, 0, stream>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<
              T, block,
              framework::DataLayout::kNCHW><<<grid2, block, 0, stream>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      } else {
        if (d_x) {
          KeBNBackwardData<
              T, framework::DataLayout::kNHWC><<<grid1, block, 0, stream>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<
              T, block,
              framework::DataLayout::kNHWC><<<grid2, block, 0, stream>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      }
    }
  }
};

template <typename T>
class BatchNormDoubleGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto *X = ctx.Input<Tensor>("X");
    const auto *Scale = ctx.Input<Tensor>("Scale");
    const auto *dY = ctx.Input<Tensor>("DY");
    const auto *Saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *Saved_variance = ctx.Input<Tensor>("SavedVariance");
    const double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const bool is_test = ctx.Attr<bool>("is_test");

    PADDLE_ENFORCE_EQ(
        is_test, false,
        platform::errors::InvalidArgument(
            "`is_test = True` CANNOT be used in train program. If "
            "you want to use global status in pre_train model, "
            "please set `use_global_stats = True`"));

    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);

    const auto *ddX = ctx.Input<Tensor>("DDX");
    const auto *ddScale = ctx.Input<Tensor>("DDScale");
    const auto *ddBias = ctx.Input<Tensor>("DDBias");

    auto *dX = ctx.Output<Tensor>("DX");
    auto *dScale = ctx.Output<Tensor>("DScale");
    auto *ddY = ctx.Output<Tensor>("DDY");

    NormDoubleGradFunctor<platform::CUDADeviceContext, T>(
        ctx, data_layout, X, Scale, dY, Saved_mean, Saved_variance, epsilon,
        use_global_stats, ddX, ddScale, ddBias, dX, dScale, ddY);
  }
};

}  // namespace operators
}  // namespace paddle

// namespace ops = paddle::operators;
// namespace plat = paddle::platform;
// #ifdef PADDLE_WITH_HIP
// // MIOPEN do not support double
// REGISTER_OP_CUDA_KERNEL(
//     batch_norm, ops::BatchNormKernel<plat::CUDADeviceContext, float>,
//     ops::BatchNormKernel<plat::CUDADeviceContext, plat::float16>);
// REGISTER_OP_CUDA_KERNEL(
//     batch_norm_grad, ops::BatchNormGradKernel<plat::CUDADeviceContext, float>,
//     ops::BatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
// REGISTER_OP_CUDA_KERNEL(
//     batch_norm_grad_grad,
//     ops::BatchNormDoubleGradKernel<plat::CUDADeviceContext, float>);
// #else
// REGISTER_OP_CUDA_KERNEL(
//     batch_norm, ops::BatchNormKernel<plat::CUDADeviceContext, float>,
//     ops::BatchNormKernel<plat::CUDADeviceContext, double>,
//     ops::BatchNormKernel<plat::CUDADeviceContext, plat::float16>);
// REGISTER_OP_CUDA_KERNEL(
//     batch_norm_grad, ops::BatchNormGradKernel<plat::CUDADeviceContext, float>,
//     ops::BatchNormGradKernel<plat::CUDADeviceContext, double>,
//     ops::BatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
// REGISTER_OP_CUDA_KERNEL(
//     batch_norm_grad_grad,
//     ops::BatchNormDoubleGradKernel<plat::CUDADeviceContext, float>,
//     ops::BatchNormDoubleGradKernel<plat::CUDADeviceContext, double>);
// #endif
