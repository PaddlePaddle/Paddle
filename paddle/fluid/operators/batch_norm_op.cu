#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"

DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
class BatchNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const bool trainable_stats = ctx.Attr<bool>("trainable_statistics");
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);

    bool test_mode = is_test && (!trainable_stats);

    // Get the size for each dimension.
    // NCHW [batch_size, in_channels, in_height, in_width]
    const auto *x = ctx.Input<Tensor>("X");
    const auto &x_dims = x->dims();
    PADDLE_ENFORCE_EQ(
        x_dims.size() >= 2 && x_dims.size() <= 5, true,
        platform::errors::InvalidArgument(
            "The size of input's dimensions should be between 2 and 5"
            "But received: the size of input's dimensions is [%d]",
            x_dims.size()));

    auto *y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    auto dtype = platform::CudnnDataType<T>::type;
    const bool fast_nhwc_batch_norm =
        test_mode ||
        (dtype == HIPDNN_DATA_HALF && FLAGS_cudnn_batchnorm_spatial_persistent);

    auto compute_format =
        fast_nhwc_batch_norm && data_layout == DataLayout::kNHWC
            ? DataLayout::kNHWC
            : DataLayout::kNCHW;

    Tensor transformed_x(x->type());
    Tensor transformed_y(y->type());
    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW && x_dims.size() > 2) {
      VLOG(3) << "Transform input tensor from NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                           &transformed_x);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                          &transformed_x);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, y,
                                                           &transformed_y);
    } else {
      transformed_x.ShareDataWith(*x);
      transformed_y.ShareDataWith(*y);
    }

    // ------------------- cudnn descriptors ---------------------
    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t bn_param_desc_;
    hipdnnBatchNormMode_t mode_;

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));

    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
#if CUDNN_VERSION_MIN(7, 0, 0)
    if (FLAGS_cudnn_batchnorm_spatial_persistent) {
      mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
    } else {
      mode_ = HIPDNN_BATCHNORM_SPATIAL;
    }
#else
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
#endif

    VLOG(3) << "Setting descriptors.";
    std::vector<int> dims;
    std::vector<int> strides;
    if (compute_format == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * D * C, 1, W * D * C, D * C, C};
    }
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
    // Note: PERSISTENT not implemented for inference
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDeriveBNTensorDescriptor(
            bn_param_desc_, data_desc_,
            test_mode ? HIPDNN_BATCHNORM_SPATIAL : mode_));

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    auto handle = dev_ctx.cudnn_handle();

    // Now, depending on whether we are running test or not, we have two paths.
    if (test_mode || use_global_stats) {
      // only when test we use input to do computation.
      const auto *est_mean = ctx.Input<Tensor>("Mean");
      const auto *est_var = ctx.Input<Tensor>("Variance");
      // Run inference mode.
      PADDLE_ENFORCE_EQ(
          est_mean->dims().size(), 1UL,
          platform::errors::InvalidArgument(
              "The size of mean's dimensions must equal to 1."
              "But received: the size of mean's dimensions mean is [%d],"
              "the dimensions of mean is [%s].",
              est_mean->dims().size(), est_mean->dims()));
      PADDLE_ENFORCE_EQ(
          est_var->dims().size(), 1UL,
          platform::errors::InvalidArgument(
              "The size of variance's dimensions must equal to 1."
              "But received: the size of variance's dimensions is [%d],"
              "the dimensions of variance is [%s].",
              est_var->dims().size(), est_var->dims()));
      PADDLE_ENFORCE_EQ(
          est_mean->dims()[0], C,
          platform::errors::InvalidArgument(
              "The first dimension of mean must equal to the number of "
              "Channels, which is [%d]. But received: the first dimension"
              "of mean is [%d], the dimensions of mean is [%s].",
              C, est_mean->dims()[0], est_mean->dims()));
      PADDLE_ENFORCE_EQ(
          est_var->dims()[0], C,
          platform::errors::InvalidArgument(
              "The first dimension of variance must equal to the number"
              "of Channels, which is [%d]. But received: the first dimension of"
              "variance is [%d], the dimensions of variance is [%s].",
              C, est_var->dims()[0], est_var->dims()));

      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnBatchNormalizationForwardInference(
              handle,
              // Note: PERSISTENT not implemented for inference
              HIPDNN_BATCHNORM_SPATIAL, CudnnDataType<T>::kOne(),
              CudnnDataType<T>::kZero(), data_desc_,
              transformed_x.template data<T>(), data_desc_,
              transformed_y.template mutable_data<T>(ctx.GetPlace()),
              bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
              bias->template data<BatchNormParamType<T>>(),
              est_mean->template data<BatchNormParamType<T>>(),
              est_var->template data<BatchNormParamType<T>>(), epsilon));
    } else {
      // if MomentumTensor is set, use MomentumTensor value, momentum
      // is only used in this training branch
      if (ctx.HasInput("MomentumTensor")) {
        const auto *mom_tensor = ctx.Input<Tensor>("MomentumTensor");
        Tensor mom_cpu;
        TensorCopySync(*mom_tensor, platform::CPUPlace(), &mom_cpu);
        momentum = mom_cpu.data<float>()[0];
      }

      // Run training mode.
      // obtain running mean and running inv var, and see if we need to
      // initialize them.

      auto *mean_out = ctx.Output<Tensor>("MeanOut");
      auto *variance_out = ctx.Output<Tensor>("VarianceOut");
      mean_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      variance_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

      auto *saved_mean = ctx.Output<Tensor>("SavedMean");
      auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
      saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
          functor;
      functor(dev_ctx, saved_mean, static_cast<BatchNormParamType<T>>(0));
      functor(dev_ctx, saved_variance, static_cast<BatchNormParamType<T>>(0));

      if ((N * H * W * D) == 1) {
        // Only 1 element in normalization dimension,
        // skip the batch norm calculation, let y = x.
        framework::TensorCopy(*x, ctx.GetPlace(), y);
      } else {
        double this_factor = 1. - momentum;

        bool called = false;
#if CUDNN_VERSION_MIN(7, 4, 1)
        if (compute_format == DataLayout::kNHWC) {
          called = true;
          size_t workspace_size = 0;
          size_t reserve_space_size = 0;
          void *reserve_space_ptr = nullptr;
          void *workspace_ptr = nullptr;
          Tensor workspace_tensor;
          // Create reserve space and workspace for batch norm.
          // Create tensor for each batchnorm op, it will be used in the
          // backward. Thus this tensor shouldn't be temp.
          auto *reserve_space = ctx.Output<Tensor>("ReserveSpace");
          PADDLE_ENFORCE_NOT_NULL(
              reserve_space,
              platform::errors::NotFound(
                  "The argument ReserveSpace of batch_norm op is not found."));

          // --------------- cudnn batchnorm workspace ---------------
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::
                  cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
                      /*handle=*/handle,
                      /*mode=*/mode_,
                      /*bnIps=*/CUDNN_BATCHNORM_OPS_BN,
                      /*xDesc=*/data_desc_,
                      /*zDesc=*/nullptr,
                      /*yDesc=*/data_desc_,
                      /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                      /*activationDesc=*/nullptr,
                      /*sizeInBytes=*/&workspace_size));

          // -------------- cudnn batchnorm reserve space --------------
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::
                  cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
                      /*handle=*/handle,
                      /*mode=*/mode_,
                      /*bnOps=*/CUDNN_BATCHNORM_OPS_BN,
                      /*activationDesc=*/nullptr,
                      /*xDesc=*/data_desc_,
                      /*sizeInBytes=*/&reserve_space_size));

          reserve_space_ptr = reserve_space->mutable_data(
              ctx.GetPlace(), transformed_x.type(), reserve_space_size);
          workspace_ptr = workspace_tensor.mutable_data(
              ctx.GetPlace(), transformed_x.type(), workspace_size);
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::cudnnBatchNormalizationForwardTrainingEx(
                  handle, mode_, CUDNN_BATCHNORM_OPS_BN,
                  CudnnDataType<T>::kOne(), CudnnDataType<T>::kZero(),
                  data_desc_, transformed_x.template data<T>(), nullptr,
                  nullptr, data_desc_, transformed_y.template data<T>(),
                  bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
                  bias->template data<BatchNormParamType<T>>(), this_factor,
                  mean_out->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  variance_out->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  epsilon,
                  saved_mean->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  saved_variance->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  nullptr, workspace_ptr, workspace_size, reserve_space_ptr,
                  reserve_space_size));
        }
#endif
        if (!called) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnBatchNormalizationForwardTraining(
                  handle, mode_, CudnnDataType<T>::kOne(),
                  CudnnDataType<T>::kZero(), data_desc_,
                  transformed_x.template data<T>(), data_desc_,
                  transformed_y.template mutable_data<T>(ctx.GetPlace()),
                  bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
                  bias->template data<BatchNormParamType<T>>(), this_factor,
                  mean_out->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  variance_out->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  epsilon,
                  saved_mean->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  saved_variance->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace())));
        }
      }
    }

    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW && x_dims.size() > 2) {
      VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
      TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
          ctx, &transformed_y, y);
    }
    // clean when exit.
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
  }
};

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void KeBNBackwardScaleBias(
    const T *dy, const T *x, const BatchNormParamType<T> *mean,
    const BatchNormParamType<T> *variance, const double epsilon, const int N,
    const int C, const int HxW, BatchNormParamType<T> *dscale,
    BatchNormParamType<T> *dbias) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ds_storage;
  __shared__ typename BlockReduce::TempStorage db_storage;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> ds_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> db_sum = static_cast<BatchNormParamType<T>>(0);

    BatchNormParamType<T> inv_var_i = 1.0 / sqrt(variance[i] + epsilon);
    BatchNormParamType<T> mean_i = mean[i];
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      ds_sum += static_cast<BatchNormParamType<T>>(dy[index]) *
                (static_cast<BatchNormParamType<T>>(x[index]) - mean_i);
      db_sum += static_cast<BatchNormParamType<T>>(dy[index]);
    }
    ds_sum = BlockReduce(ds_storage).Reduce(ds_sum, hipcub::Sum());
    db_sum = BlockReduce(db_storage).Reduce(db_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      dscale[i] = ds_sum * inv_var_i;
      dbias[i] = db_sum;
    }
    __syncthreads();
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeBNBackwardData(const T *dy,
                                        const BatchNormParamType<T> *scale,
                                        const BatchNormParamType<T> *variance,
                                        const double epsilon, const int C,
                                        const int HxW, const int num, T *dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? i / HxW % C : i % C;
    BatchNormParamType<T> inv_var = 1.0 / sqrt(variance[c] + epsilon);
    dx[i] = static_cast<T>(static_cast<BatchNormParamType<T>>(dy[i]) *
                           scale[c] * inv_var);
  }
}

template <typename T>
static __global__ void KeBNRestoreData(const framework::DataLayout layout, T *x,
                                       const BatchNormParamType<T> *scale,
                                       const BatchNormParamType<T> *bias,
                                       const BatchNormParamType<T> *mean,
                                       const BatchNormParamType<T> *variance,
                                       double epsilon, int C, int M,
                                       const int num, const T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? (i / M) % C : i % C;
    auto y_i = static_cast<BatchNormParamType<T>>(y[i]);
    auto x_i = (y_i - bias[c]) / scale[c] / variance[c] + mean[c];
    x[i] = static_cast<T>(x_i);
  }
}

template <typename T>
class InplaceHelper {
 public:
  void operator()(const framework::DataLayout layout, T *x,
                  const BatchNormParamType<T> *scale,
                  const BatchNormParamType<T> *bias,
                  const BatchNormParamType<T> *mean,
                  const BatchNormParamType<T> *variance, double epsilon, int C,
                  int M, const int num, const T *y, int grid2, const int block,
                  const hipStream_t &stream) {
    PADDLE_ENFORCE_EQ(x, y, platform::errors::InvalidArgument(
                                "X and Y should be inplaced in inplace mode"));
    KeBNRestoreData<<<grid2, block, 0, stream>>>(
        layout, x, scale, bias, mean, variance, epsilon, C, M, num, y);
  }
};

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void BNBackwardData(const T *dy,
                                      const BatchNormParamType<T> *scale,
                                      const BatchNormParamType<T> *mean,
                                      const T *x,
                                      const BatchNormParamType<T> *variance,
                                      const int C, const int N, const int HxW,
                                      T *dx) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> inv_var_i = variance[i];
    BatchNormParamType<T> mean_i = mean[i];
    BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> dy_x_sub_mean_sum =
        static_cast<BatchNormParamType<T>>(0);
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      BatchNormParamType<T> dy_i =
          static_cast<BatchNormParamType<T>>(dy[index]);
      dy_sum += dy_i;
      dy_x_sub_mean_sum +=
          dy_i * (static_cast<BatchNormParamType<T>>(x[index]) - mean_i);
    }

    dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
    dy_x_sub_mean_sum = BlockReduce(dy_x_sub_mean_storage)
                            .Reduce(dy_x_sub_mean_sum, hipcub::Sum());

    if (threadIdx.x == 0) {
      dy_sum_val = dy_sum;
      dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
    }
    __syncthreads();

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      dx[index] =
          (static_cast<BatchNormParamType<T>>(dy[index]) -
           dy_sum_val / static_cast<BatchNormParamType<T>>(inner_size) -
           (static_cast<BatchNormParamType<T>>(x[index]) - mean_i) *
               dy_x_sub_mean_sum_val * inv_var_i * inv_var_i / inner_size) *
          scale[i] * inv_var_i;
    }
  }
}

template <typename T>
class BatchNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");

    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    // batch_norm with inplace as false will take X as grad input, which
    // is same as cuDNN batch_norm backward calculation, batch_norm
    // with inplace as true only take Y as input and X should be calculate
    // by inverse operation of batch_norm on Y
    const Tensor *x;
    bool is_inplace;
    if (ctx.HasInput("Y")) {
      x = ctx.Input<Tensor>("Y");
      is_inplace = true;
      PADDLE_ENFORCE_EQ(d_x, d_y,
                        platform::errors::InvalidArgument(
                            "X@GRAD and Y@GRAD not inplace in inplace mode"));
    } else {
      x = ctx.Input<Tensor>("X");
      is_inplace = false;
      PADDLE_ENFORCE_NE(d_x, d_y,
                        platform::errors::InvalidArgument(
                            "X@GRAD and Y@GRAD inplaced in non-inplace mode"));
    }

    const bool is_test = ctx.Attr<bool>("is_test");
    PADDLE_ENFORCE_EQ(
        is_test, false,
        platform::errors::InvalidArgument(
            "`is_test = True` CANNOT be used in train program. If "
            "you want to use global status in pre_train model, "
            "please set `use_global_stats = True`"));

    const auto &x_dims = x->dims();

    PADDLE_ENFORCE_EQ(
        x_dims.size() >= 2 && x_dims.size() <= 5, true,
        platform::errors::InvalidArgument(
            "The size of input's dimensions should be between 2 and 5."
            "But received: the size of input's dimensions is [%d],"
            "the dimensions of input is [%s]",
            x_dims.size(), x_dims));
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    // init output
    d_x->mutable_data<T>(ctx.GetPlace());

    if (d_scale && d_bias) {
      d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(
        scale->dims().size(), 1UL,
        platform::errors::InvalidArgument(
            "The size of scale's dimensions must equal to 1. But received: "
            "the size of scale's dimensions is [%d], the dimensions of scale "
            "is [%s].",
            scale->dims().size(), scale->dims()));
    PADDLE_ENFORCE_EQ(
        scale->dims()[0], C,
        platform::errors::InvalidArgument(
            "The first dimension of scale must equal to Channels[%d]. But "
            "received: the first dimension of scale is [%d]",
            C, scale->dims()[0]));

    auto dtype = platform::CudnnDataType<T>::type;
    const auto *reserve_space = ctx.Input<Tensor>("ReserveSpace");
    const bool fast_nhwc_batch_norm =
        dtype == HIPDNN_DATA_HALF && FLAGS_cudnn_batchnorm_spatial_persistent &&
        reserve_space != nullptr;
    auto compute_format =
        fast_nhwc_batch_norm && data_layout == DataLayout::kNHWC
            ? DataLayout::kNHWC
            : DataLayout::kNCHW;

    Tensor transformed_x(x->type());
    Tensor transformed_d_y(d_y->type());
    Tensor transformed_d_x(d_x->type());
    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW) {
      VLOG(3) << "Transform input tensor from NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                           &transformed_x);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                          &transformed_x);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_y,
                                                           &transformed_d_y);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_y,
                                                          &transformed_d_y);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_x,
                                                           &transformed_d_x);
    } else {
      transformed_x.ShareDataWith(*x);
      transformed_d_y.ShareDataWith(*d_y);
      transformed_d_x.ShareDataWith(*d_x);
    }

    std::vector<int> dims;
    std::vector<int> strides;
    if (compute_format == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * C * D, 1, W * D * C, D * C, C};
    }

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    const int num = transformed_x.numel();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    int grid1 = (num + block - 1) / block;
    int grid2 = std::min(C, max_blocks);
    auto stream = dev_ctx.stream();
    InplaceHelper<T> inplace_functor;

    if (!use_global_stats) {
      if ((N * H * W * D) == 1) {
        framework::TensorCopy(*d_y, ctx.GetPlace(), d_x);
        math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
            functor;
        functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
        functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
        return;
      }

      // ------------------- cudnn descriptors ---------------------
      hipdnnTensorDescriptor_t data_desc_;
      hipdnnTensorDescriptor_t bn_param_desc_;
      hipdnnBatchNormMode_t mode_;

      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
      if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
        LOG(ERROR) << "Provided epsilon is smaller than "
                   << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                   << "HIPDNN_BN_MIN_EPSILON instead.";
      }
      epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
#if CUDNN_VERSION_MIN(7, 0, 0)
      if (FLAGS_cudnn_batchnorm_spatial_persistent) {
        mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
      } else {
        mode_ = HIPDNN_BATCHNORM_SPATIAL;
      }
#else
      mode_ = HIPDNN_BATCHNORM_SPATIAL;
#endif

      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_, CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDeriveBNTensorDescriptor(bn_param_desc_,
                                                           data_desc_, mode_));

      const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
      const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
      const auto *saved_mean_data =
          saved_mean->template data<BatchNormParamType<T>>();
      const auto *saved_var_data =
          saved_var->template data<BatchNormParamType<T>>();

      if (is_inplace) {
        inplace_functor(compute_format, transformed_x.data<T>(),
                        scale->template data<BatchNormParamType<T>>(),
                        bias->template data<BatchNormParamType<T>>(),
                        saved_mean_data, saved_var_data, epsilon, C, H * W * D,
                        num, transformed_x.data<T>(), grid2, block, stream);
      }

      if (d_scale && d_bias) {
        bool called = false;
#if CUDNN_VERSION_MIN(7, 4, 1)
        if (compute_format == DataLayout::kNHWC) {
          called = true;
          size_t workspace_size = 0;
          void *workspace_ptr = nullptr;
          Tensor workspace_tensor;
          auto reserve_space_size = reserve_space->memory_size();
          // --------------- cudnn batchnorm workspace ---------------
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::
                  cudnnGetBatchNormalizationBackwardExWorkspaceSize(
                      /*handle=*/dev_ctx.cudnn_handle(),
                      /*mode=*/mode_,
                      /*bnIps=*/CUDNN_BATCHNORM_OPS_BN,
                      /*xDesc=*/data_desc_,
                      /*yDesc=*/data_desc_,
                      /*dyDesc=*/data_desc_,
                      /*dzDesc=*/nullptr,
                      /*dxDesc=*/data_desc_,
                      /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                      /*activationDesc=*/nullptr,
                      /*sizeInBytes=*/&workspace_size));

          workspace_ptr = workspace_tensor.mutable_data(
              ctx.GetPlace(), transformed_x.type(), workspace_size);

          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::cudnnBatchNormalizationBackwardEx(
                  /*handle=*/dev_ctx.cudnn_handle(),
                  /*mode=*/mode_,
                  /*bnOps=*/CUDNN_BATCHNORM_OPS_BN,
                  /*alphaDataDiff=*/CudnnDataType<T>::kOne(),
                  /*betaDataDiff=*/CudnnDataType<T>::kZero(),
                  /*alphaParamDiff=*/CudnnDataType<T>::kOne(),
                  /*betaParamDiff=*/CudnnDataType<T>::kZero(),
                  /*xDesc=*/data_desc_,
                  /*xData=*/transformed_x.template data<T>(),
                  /*yDesc=*/nullptr,
                  /*yData=*/nullptr,
                  /*dyDesc=*/data_desc_,
                  /*dyData=*/transformed_d_y.template data<T>(),
                  /*dzDesc=*/nullptr,
                  /*dzData=*/nullptr,
                  /*dxDesc=*/data_desc_,
                  /*dxData=*/transformed_d_x.template mutable_data<T>(
                      ctx.GetPlace()),
                  /*dBnScaleBiasDesc=*/bn_param_desc_,
                  /*bnScaleData=*/scale->template data<BatchNormParamType<T>>(),
                  /*bnBiasData=*/nullptr,
                  /*dBnScaleData=*/d_scale
                      ->template mutable_data<BatchNormParamType<T>>(
                          ctx.GetPlace()),
                  /*dBnBiasData=*/d_bias
                      ->template mutable_data<BatchNormParamType<T>>(
                          ctx.GetPlace()),
                  /*epsilon=*/epsilon,
                  /*savedMean=*/saved_mean_data,
                  /*savedInvVariance=*/saved_var_data,
                  /*activationDesc=*/nullptr,
                  /*workspace=*/workspace_ptr,
                  /*workSpaceSizeInBytes=*/workspace_size,
                  /*reserveSpace=*/const_cast<T *>(
                      reserve_space->template data<T>()),
                  /*reserveSpaceSizeInBytes=*/reserve_space_size));
        }
#endif
        if (!called) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnBatchNormalizationBackward(
                  dev_ctx.cudnn_handle(), mode_, CudnnDataType<T>::kOne(),
                  CudnnDataType<T>::kZero(), CudnnDataType<T>::kOne(),
                  CudnnDataType<T>::kZero(), data_desc_,
                  transformed_x.template data<T>(), data_desc_,
                  transformed_d_y.template data<T>(), data_desc_,
                  transformed_d_x.template mutable_data<T>(ctx.GetPlace()),
                  bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
                  d_scale->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  d_bias->template mutable_data<BatchNormParamType<T>>(
                      ctx.GetPlace()),
                  epsilon, saved_mean_data, saved_var_data));
        }

        if (data_layout == DataLayout::kNHWC &&
            compute_format == DataLayout::kNCHW) {
          VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
          TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
              ctx, &transformed_d_x, d_x);
        }
      } else {
        if (compute_format == DataLayout::kNCHW) {
          if (d_x) {
            BNBackwardData<T, block, framework::DataLayout::kNCHW><<<
                grid2, block, 0, dev_ctx.stream()>>>(
                d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
                saved_mean_data, x->data<T>(), saved_var_data, C, N, H * W * D,
                d_x->data<T>());
          }
        } else {
          if (d_x) {
            BNBackwardData<T, block, framework::DataLayout::kNHWC><<<
                grid2, block, 0, dev_ctx.stream()>>>(
                d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
                saved_mean_data, x->data<T>(), saved_var_data, C, N, H * W * D,
                d_x->data<T>());
          }
        }
      }

      // clean when exit.
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
    } else {
      const auto *running_mean = ctx.Input<Tensor>("Mean");
      const auto *running_var = ctx.Input<Tensor>("Variance");

      const auto *running_mean_data =
          running_mean->template data<BatchNormParamType<T>>();
      const auto *running_var_data =
          running_var->template data<BatchNormParamType<T>>();

      if (is_inplace) {
        auto px = *x;
        inplace_functor(data_layout, px.mutable_data<T>(ctx.GetPlace()),
                        scale->template data<BatchNormParamType<T>>(),
                        bias->template data<BatchNormParamType<T>>(),
                        running_mean_data, running_var_data, epsilon, C,
                        H * W * D, num, x->data<T>(), grid2, block, stream);
      }

      if (compute_format == DataLayout::kNCHW) {
        if (d_x) {
          KeBNBackwardData<
              T, framework::DataLayout::kNCHW><<<grid1, block, 0, stream>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<
              T, block,
              framework::DataLayout::kNCHW><<<grid2, block, 0, stream>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      } else {
        if (d_x) {
          KeBNBackwardData<
              T, framework::DataLayout::kNHWC><<<grid1, block, 0, stream>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<
              T, block,
              framework::DataLayout::kNHWC><<<grid2, block, 0, stream>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    batch_norm, ops::BatchNormKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormKernel<plat::CUDADeviceContext, double>,
    ops::BatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    batch_norm_grad, ops::BatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, double>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
