#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/framework/data_layout.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/batch_norm_miopen_helper.h"
#else
#include "paddle/fluid/operators/batch_norm_cudnn_helper.h"
#endif
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/norm_utils.cu.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
class BatchNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const bool trainable_stats = ctx.Attr<bool>("trainable_statistics");
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);

    // Get the size for each dimension.
    // NCHW [batch_size, in_channels, in_height, in_width]
    const auto *x = ctx.Input<Tensor>("X");
    const auto &x_dims = x->dims();
    PADDLE_ENFORCE_EQ(
        x_dims.size() >= 2 && x_dims.size() <= 5, true,
        platform::errors::InvalidArgument(
            "The size of input's dimensions should be between 2 and 5"
            "But received: the size of input's dimensions is [%d]",
            x_dims.size()));

    auto *y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    bool test_mode = is_test && (!trainable_stats);
    auto compute_format = GetComputeFormat<T>(data_layout, test_mode);

    Tensor transformed_x(x->type());
    Tensor transformed_y(y->type());
    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW && x_dims.size() > 2) {
      VLOG(3) << "Transform input tensor from NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                           &transformed_x);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                          &transformed_x);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, y,
                                                           &transformed_y);
    } else {
      transformed_x.ShareDataWith(*x);
      transformed_y.ShareDataWith(*y);
    }

    PADDLE_ENFORCE_GT(epsilon, HIPDNN_BN_MIN_EPSILON - FLT_EPSILON,
                      platform::errors::InvalidArgument(
                          "Provided epsilon is expected to be greater than "
                          "HIPDNN_BN_MIN_EPSILON. But recieved epsilon is %E, "
                          "HIPDNN_BN_MIN_EPSILON is %E.",
                          epsilon, HIPDNN_BN_MIN_EPSILON));
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    std::vector<int> dims;
    std::vector<int> strides;
    if (compute_format == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * D * C, 1, W * D * C, D * C, C};
    }

    // cudnn descriptors
    VLOG(3) << "Setting cudnn/miopen descriptors.";
    BatchNormMode mode = GetBatchNormMode(test_mode);
    BatchNormWrapper<T> bn_wrapper(x_dims.size() > 3 ? x_dims.size() : 4, dims,
                                   strides, mode, epsilon, false, false);

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    // Depending on whether we are running test or not, we have two paths.
    // It is training mode when it's not reference AND not using pre-trained
    // model.
    bool training = !(test_mode || use_global_stats);
    if (!training) {
      // only when test we use input to do computation.
      const auto *est_mean = ctx.Input<Tensor>("Mean");
      const auto *est_var = ctx.Input<Tensor>("Variance");
      // Run inference mode.
      PADDLE_ENFORCE_EQ(
          est_mean->dims().size(), 1UL,
          platform::errors::InvalidArgument(
              "The size of mean's dimensions must equal to 1."
              "But received: the size of mean's dimensions mean is [%d],"
              "the dimensions of mean is [%s].",
              est_mean->dims().size(), est_mean->dims()));
      PADDLE_ENFORCE_EQ(
          est_var->dims().size(), 1UL,
          platform::errors::InvalidArgument(
              "The size of variance's dimensions must equal to 1."
              "But received: the size of variance's dimensions is [%d],"
              "the dimensions of variance is [%s].",
              est_var->dims().size(), est_var->dims()));
      PADDLE_ENFORCE_EQ(
          est_mean->dims()[0], C,
          platform::errors::InvalidArgument(
              "The first dimension of mean must equal to the number of "
              "Channels, which is [%d]. But received: the first dimension"
              "of mean is [%d], the dimensions of mean is [%s].",
              C, est_mean->dims()[0], est_mean->dims()));
      PADDLE_ENFORCE_EQ(
          est_var->dims()[0], C,
          platform::errors::InvalidArgument(
              "The first dimension of variance must equal to the number"
              "of Channels, which is [%d]. But received: the first dimension of"
              "variance is [%d], the dimensions of variance is [%s].",
              C, est_var->dims()[0], est_var->dims()));

      bn_wrapper.Infer(dev_ctx, transformed_x, *scale, *bias, *est_mean,
                       *est_var, &transformed_y);
    } else {
      // If MomentumTensor is set, use MomentumTensor value, momentum
      // is only used in this training branch
      if (ctx.HasInput("MomentumTensor")) {
        const auto *mom_tensor = ctx.Input<Tensor>("MomentumTensor");
        Tensor mom_cpu;
        TensorCopySync(*mom_tensor, platform::CPUPlace(), &mom_cpu);
        momentum = mom_cpu.data<float>()[0];
      }

      if ((N * H * W * D) == 1) {
        // Only 1 element in normalization dimension,
        // skip the batch norm calculation, let y = x.
        framework::TensorCopy(*x, ctx.GetPlace(), y);
      } else {
        double this_factor = 1. - momentum;

        auto *mean_out = ctx.Output<Tensor>("MeanOut");
        auto *variance_out = ctx.Output<Tensor>("VarianceOut");
        auto *saved_mean = ctx.Output<Tensor>("SavedMean");
        auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
        auto *reserve_space = ctx.Output<Tensor>("ReserveSpace");

        saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
        saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
        math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
            set_zero;
        set_zero(dev_ctx, saved_mean, static_cast<BatchNormParamType<T>>(0));
        set_zero(dev_ctx, saved_variance,
                 static_cast<BatchNormParamType<T>>(0));

        bn_wrapper.TrainForward(dev_ctx, transformed_x, *scale, *bias,
                                &transformed_y, mean_out, variance_out,
                                saved_mean, saved_variance, reserve_space,
                                this_factor);
      }
    }

    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW && x_dims.size() > 2) {
      VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
      TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
          ctx, &transformed_y, y);
    }
  }
};

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void KeBNBackwardScaleBias(
    const T *dy, const T *x, const BatchNormParamType<T> *mean,
    const BatchNormParamType<T> *variance, const double epsilon, const int N,
    const int C, const int HxW, BatchNormParamType<T> *dscale,
    BatchNormParamType<T> *dbias) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ds_storage;
  __shared__ typename BlockReduce::TempStorage db_storage;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> ds_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> db_sum = static_cast<BatchNormParamType<T>>(0);

    BatchNormParamType<T> inv_var_i = 1.0 / sqrt(variance[i] + epsilon);
    BatchNormParamType<T> mean_i = mean[i];
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      ds_sum += static_cast<BatchNormParamType<T>>(dy[index]) *
                (static_cast<BatchNormParamType<T>>(x[index]) - mean_i);
      db_sum += static_cast<BatchNormParamType<T>>(dy[index]);
    }
    ds_sum = BlockReduce(ds_storage).Reduce(ds_sum, hipcub::Sum());
    db_sum = BlockReduce(db_storage).Reduce(db_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      dscale[i] = ds_sum * inv_var_i;
      dbias[i] = db_sum;
    }
    __syncthreads();
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeBNBackwardData(const T *dy,
                                        const BatchNormParamType<T> *scale,
                                        const BatchNormParamType<T> *variance,
                                        const double epsilon, const int C,
                                        const int HxW, const int num, T *dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? i / HxW % C : i % C;
    BatchNormParamType<T> inv_var = 1.0 / sqrt(variance[c] + epsilon);
    dx[i] = static_cast<T>(static_cast<BatchNormParamType<T>>(dy[i]) *
                           scale[c] * inv_var);
  }
}

template <typename T>
static __global__ void KeBNRestoreData(const framework::DataLayout layout, T *x,
                                       const BatchNormParamType<T> *scale,
                                       const BatchNormParamType<T> *bias,
                                       const BatchNormParamType<T> *mean,
                                       const BatchNormParamType<T> *variance,
                                       double epsilon, int C, int M,
                                       const int num, const T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? (i / M) % C : i % C;
    auto y_i = static_cast<BatchNormParamType<T>>(y[i]);
    auto x_i = (y_i - bias[c]) / scale[c] / variance[c] + mean[c];
    x[i] = static_cast<T>(x_i);
  }
}

template <typename T>
class InplaceHelper {
 public:
  void operator()(const framework::DataLayout layout, T *x,
                  const BatchNormParamType<T> *scale,
                  const BatchNormParamType<T> *bias,
                  const BatchNormParamType<T> *mean,
                  const BatchNormParamType<T> *variance, double epsilon, int C,
                  int M, const int num, const T *y, int grid2, const int block,
                  const gpuStream_t &stream) {
    PADDLE_ENFORCE_EQ(x, y, platform::errors::InvalidArgument(
                                "X and Y should be inplaced in inplace mode"));
    KeBNRestoreData<<<grid2, block, 0, stream>>>(
        layout, x, scale, bias, mean, variance, epsilon, C, M, num, y);
  }
};

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void BNBackwardData(const T *dy,
                                      const BatchNormParamType<T> *scale,
                                      const BatchNormParamType<T> *mean,
                                      const T *x,
                                      const BatchNormParamType<T> *variance,
                                      const int C, const int N, const int HxW,
                                      T *dx) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> inv_var_i = variance[i];
    BatchNormParamType<T> mean_i = mean[i];
    BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> dy_x_sub_mean_sum =
        static_cast<BatchNormParamType<T>>(0);
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      BatchNormParamType<T> dy_i =
          static_cast<BatchNormParamType<T>>(dy[index]);
      dy_sum += dy_i;
      dy_x_sub_mean_sum +=
          dy_i * (static_cast<BatchNormParamType<T>>(x[index]) - mean_i);
    }

    dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
    dy_x_sub_mean_sum = BlockReduce(dy_x_sub_mean_storage)
                            .Reduce(dy_x_sub_mean_sum, hipcub::Sum());

    if (threadIdx.x == 0) {
      dy_sum_val = dy_sum;
      dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
    }
    __syncthreads();

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      dx[index] =
          (static_cast<BatchNormParamType<T>>(dy[index]) -
           dy_sum_val / static_cast<BatchNormParamType<T>>(inner_size) -
           (static_cast<BatchNormParamType<T>>(x[index]) - mean_i) *
               dy_x_sub_mean_sum_val * inv_var_i * inv_var_i / inner_size) *
          scale[i] * inv_var_i;
    }
  }
}

template <typename T>
class BatchNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");

    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    // batch_norm with inplace as false will take X as grad input, which
    // is same as cuDNN batch_norm backward calculation, batch_norm
    // with inplace as true only take Y as input and X should be calculate
    // by inverse operation of batch_norm on Y
    const Tensor *x;
    bool is_inplace;
    if (ctx.HasInput("Y")) {
      x = ctx.Input<Tensor>("Y");
      is_inplace = true;
      PADDLE_ENFORCE_EQ(d_x, d_y,
                        platform::errors::InvalidArgument(
                            "X@GRAD and Y@GRAD not inplace in inplace mode"));
    } else {
      x = ctx.Input<Tensor>("X");
      is_inplace = false;
      PADDLE_ENFORCE_NE(d_x, d_y,
                        platform::errors::InvalidArgument(
                            "X@GRAD and Y@GRAD inplaced in non-inplace mode"));
    }

    const bool is_test = ctx.Attr<bool>("is_test");
    PADDLE_ENFORCE_EQ(
        is_test, false,
        platform::errors::InvalidArgument(
            "`is_test = True` CANNOT be used in train program. If "
            "you want to use global status in pre_train model, "
            "please set `use_global_stats = True`"));

    const auto &x_dims = x->dims();

    PADDLE_ENFORCE_EQ(
        x_dims.size() >= 2 && x_dims.size() <= 5, true,
        platform::errors::InvalidArgument(
            "The size of input's dimensions should be between 2 and 5."
            "But received: the size of input's dimensions is [%d],"
            "the dimensions of input is [%s]",
            x_dims.size(), x_dims));
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    // init output
    d_x->mutable_data<T>(ctx.GetPlace());

    if (d_scale && d_bias) {
      d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(
        scale->dims().size(), 1UL,
        platform::errors::InvalidArgument(
            "The size of scale's dimensions must equal to 1. But received: "
            "the size of scale's dimensions is [%d], the dimensions of scale "
            "is [%s].",
            scale->dims().size(), scale->dims()));
    PADDLE_ENFORCE_EQ(
        scale->dims()[0], C,
        platform::errors::InvalidArgument(
            "The first dimension of scale must equal to Channels[%d]. But "
            "received: the first dimension of scale is [%d]",
            C, scale->dims()[0]));

    const auto *reserve_space = ctx.Input<Tensor>("ReserveSpace");
    auto compute_format =
        GetComputeFormat<T>(data_layout, false, reserve_space != nullptr);

    Tensor transformed_x(x->type());
    Tensor transformed_d_y(d_y->type());
    Tensor transformed_d_x(d_x->type());
    if (data_layout == DataLayout::kNHWC &&
        compute_format == DataLayout::kNCHW) {
      VLOG(3) << "Transform input tensor from NHWC to NCHW.";
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                           &transformed_x);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, x,
                                                          &transformed_x);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_y,
                                                           &transformed_d_y);
      TransToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_y,
                                                          &transformed_d_y);
      ResizeToChannelFirst<platform::CUDADeviceContext, T>(ctx, d_x,
                                                           &transformed_d_x);
    } else {
      transformed_x.ShareDataWith(*x);
      transformed_d_y.ShareDataWith(*d_y);
      transformed_d_x.ShareDataWith(*d_x);
    }

    std::vector<int> dims;
    std::vector<int> strides;
    if (compute_format == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * C * D, 1, W * D * C, D * C, C};
    }

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    const int num = transformed_x.numel();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    int grid1 = (num + block - 1) / block;
    int grid2 = std::min(C, max_blocks);
    auto stream = dev_ctx.stream();
    InplaceHelper<T> inplace_functor;

    if (!use_global_stats) {
      if ((N * H * W * D) == 1) {
        framework::TensorCopy(*d_y, ctx.GetPlace(), d_x);
        math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
            functor;
        functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
        functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
        return;
      }

      PADDLE_ENFORCE_GT(epsilon, HIPDNN_BN_MIN_EPSILON - FLT_EPSILON,
                        platform::errors::InvalidArgument(
                            "Provided epsilon is expected to be greater than "
                            "HIPDNN_BN_MIN_EPSILON. But recieved epsilon is %E, "
                            "HIPDNN_BN_MIN_EPSILON is %E.",
                            epsilon, HIPDNN_BN_MIN_EPSILON));
      epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

      const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
      const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
      const auto *saved_mean_data =
          saved_mean->template data<BatchNormParamType<T>>();
      const auto *saved_var_data =
          saved_var->template data<BatchNormParamType<T>>();

      if (is_inplace) {
        inplace_functor(compute_format, transformed_x.data<T>(),
                        scale->template data<BatchNormParamType<T>>(),
                        bias->template data<BatchNormParamType<T>>(),
                        saved_mean_data, saved_var_data, epsilon, C, H * W * D,
                        num, transformed_x.data<T>(), grid2, block, stream);
      }

      if (d_scale && d_bias) {
        // Call CUDNN APIs
        BatchNormMode mode = GetBatchNormMode(false);
        BatchNormWrapper<T> bn_wrapper(x_dims.size() > 3 ? x_dims.size() : 4,
                                       dims, strides, mode, epsilon, false,
                                       false);

        bn_wrapper.TrainBackward(
            dev_ctx, transformed_x, transformed_d_y, *scale, *saved_mean,
            *saved_var, *reserve_space, &transformed_d_x, d_scale, d_bias);

        if (data_layout == DataLayout::kNHWC &&
            compute_format == DataLayout::kNCHW) {
          VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
          TransToChannelLast<paddle::platform::CUDADeviceContext, T>(
              ctx, &transformed_d_x, d_x);
        }
      } else {
        // This branch call CUDA kernels
        if (compute_format == DataLayout::kNCHW) {
          if (d_x) {
            BNBackwardData<T, block, framework::DataLayout::kNCHW><<<
                grid2, block, 0, dev_ctx.stream()>>>(
                d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
                saved_mean_data, x->data<T>(), saved_var_data, C, N, H * W * D,
                d_x->data<T>());
          }
        } else {
          if (d_x) {
            BNBackwardData<T, block, framework::DataLayout::kNHWC><<<
                grid2, block, 0, dev_ctx.stream()>>>(
                d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
                saved_mean_data, x->data<T>(), saved_var_data, C, N, H * W * D,
                d_x->data<T>());
          }
        }
      }
    } else {
      const auto *running_mean = ctx.Input<Tensor>("Mean");
      const auto *running_var = ctx.Input<Tensor>("Variance");

      const auto *running_mean_data =
          running_mean->template data<BatchNormParamType<T>>();
      const auto *running_var_data =
          running_var->template data<BatchNormParamType<T>>();

      if (is_inplace) {
        auto px = *x;
        inplace_functor(data_layout, px.mutable_data<T>(ctx.GetPlace()),
                        scale->template data<BatchNormParamType<T>>(),
                        bias->template data<BatchNormParamType<T>>(),
                        running_mean_data, running_var_data, epsilon, C,
                        H * W * D, num, x->data<T>(), grid2, block, stream);
      }

      if (compute_format == DataLayout::kNCHW) {
        if (d_x) {
          KeBNBackwardData<
              T, framework::DataLayout::kNCHW><<<grid1, block, 0, stream>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<
              T, block,
              framework::DataLayout::kNCHW><<<grid2, block, 0, stream>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      } else {
        if (d_x) {
          KeBNBackwardData<
              T, framework::DataLayout::kNHWC><<<grid1, block, 0, stream>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<
              T, block,
              framework::DataLayout::kNHWC><<<grid2, block, 0, stream>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      }
    }
  }
};

template <typename T>
class BatchNormDoubleGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto *X = ctx.Input<Tensor>("X");
    const auto *Scale = ctx.Input<Tensor>("Scale");
    const auto *dY = ctx.Input<Tensor>("DY");
    const auto *Saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *Saved_variance = ctx.Input<Tensor>("SavedVariance");
    const double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const bool is_test = ctx.Attr<bool>("is_test");

    PADDLE_ENFORCE_EQ(
        is_test, false,
        platform::errors::InvalidArgument(
            "`is_test = True` CANNOT be used in train program. If "
            "you want to use global status in pre_train model, "
            "please set `use_global_stats = True`"));

    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);

    const auto *ddX = ctx.Input<Tensor>("DDX");
    const auto *ddScale = ctx.Input<Tensor>("DDScale");
    const auto *ddBias = ctx.Input<Tensor>("DDBias");

    auto *dX = ctx.Output<Tensor>("DX");
    auto *dScale = ctx.Output<Tensor>("DScale");
    auto *ddY = ctx.Output<Tensor>("DDY");

    NormDoubleGradFunctor<platform::CUDADeviceContext, T>(
        ctx, data_layout, X, Scale, dY, Saved_mean, Saved_variance, epsilon,
        use_global_stats, ddX, ddScale, ddBias, dX, dScale, ddY);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    batch_norm, ops::BatchNormKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    batch_norm_grad, ops::BatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    batch_norm_grad_grad,
    ops::BatchNormDoubleGradKernel<plat::CUDADeviceContext, float>);
#else
REGISTER_OP_CUDA_KERNEL(
    batch_norm, ops::BatchNormKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormKernel<plat::CUDADeviceContext, double>,
    ops::BatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    batch_norm_grad, ops::BatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, double>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    batch_norm_grad_grad,
    ops::BatchNormDoubleGradKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormDoubleGradKernel<plat::CUDADeviceContext, double>);
#endif
