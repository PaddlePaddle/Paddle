#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"

// HIPDNN_BATCHNORM_SPATIAL_PERSISTENT in batchnorm. This mode can be faster in
// some tasks because an optimized path may be selected for HIPDNN_DATA_FLOAT
// and HIPDNN_DATA_HALF data types, compute capability 6.0 or higher. The
// reason we set it to false by default is that this mode may use scaled
// atomic integer reduction that may cause a numerical overflow for certain
// input data range.
DEFINE_bool(cudnn_batchnorm_spatial_persistent, false,
            "Whether enable HIPDNN_BATCHNORM_SPATIAL_PERSISTENT mode for cudnn "
            "batch_norm, default is False.");

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
class BatchNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);

    // Get the size for each dimension.
    // NCHW [batch_size, in_channels, in_height, in_width]
    const auto *x = ctx.Input<Tensor>("X");
    const auto &x_dims = x->dims();
    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    auto *y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    // ------------------- cudnn descriptors ---------------------
    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t bn_param_desc_;
    hipdnnBatchNormMode_t mode_;

    CUDNN_ENFORCE(platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_ENFORCE(
        platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));

    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
#if CUDNN_VERSION_MIN(7, 0, 0)
    if (FLAGS_cudnn_batchnorm_spatial_persistent) {
      mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
    } else {
      mode_ = HIPDNN_BATCHNORM_SPATIAL;
    }
#else
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
#endif

    VLOG(3) << "Setting descriptors.";
    std::vector<int> dims;
    std::vector<int> strides;
    if (data_layout == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * D * C, 1, W * D * C, D * C, C};
    }
    CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
    // Note: PERSISTENT not implemented for inference
    CUDNN_ENFORCE(platform::dynload::hipdnnDeriveBNTensorDescriptor(
        bn_param_desc_, data_desc_, is_test ? HIPDNN_BATCHNORM_SPATIAL : mode_));

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    auto handle = dev_ctx.cudnn_handle();

    // Now, depending on whether we are running test or not, we have two paths.
    if (is_test || use_global_stats) {
      // only when test we use input to do computation.
      const auto *est_mean = ctx.Input<Tensor>("Mean");
      const auto *est_var = ctx.Input<Tensor>("Variance");
      // Run inference mode.
      PADDLE_ENFORCE_EQ(est_mean->dims().size(), 1UL);
      PADDLE_ENFORCE_EQ(est_var->dims().size(), 1UL);
      PADDLE_ENFORCE_EQ(est_mean->dims()[0], C);
      PADDLE_ENFORCE_EQ(est_var->dims()[0], C);

      CUDNN_ENFORCE(platform::dynload::hipdnnBatchNormalizationForwardInference(
          handle,
          // Note: PERSISTENT not implemented for inference
          HIPDNN_BATCHNORM_SPATIAL, CudnnDataType<T>::kOne(),
          CudnnDataType<T>::kZero(), data_desc_, x->template data<T>(),
          data_desc_, y->template mutable_data<T>(ctx.GetPlace()),
          bn_param_desc_, scale->template data<BatchNormParamType<T>>(),
          bias->template data<BatchNormParamType<T>>(),
          est_mean->template data<BatchNormParamType<T>>(),
          est_var->template data<BatchNormParamType<T>>(), epsilon));
    } else {
      // Run training mode.
      // obtain running mean and running inv var, and see if we need to
      // initialize them.

      auto *mean_out = ctx.Output<Tensor>("MeanOut");
      auto *variance_out = ctx.Output<Tensor>("VarianceOut");
      mean_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      variance_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

      auto *saved_mean = ctx.Output<Tensor>("SavedMean");
      auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
      saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
          functor;
      functor(dev_ctx, saved_mean, static_cast<BatchNormParamType<T>>(0));
      functor(dev_ctx, saved_variance, static_cast<BatchNormParamType<T>>(0));

      if ((N * H * W * D) == 1) {
        LOG(WARNING) << "Only 1 element in normalization dimension, "
                     << "we skip the batch norm calculation, let y = x.";
        framework::TensorCopy(*x, ctx.GetPlace(), y);
      } else {
        double this_factor = 1. - momentum;

        CUDNN_ENFORCE(platform::dynload::hipdnnBatchNormalizationForwardTraining(
            handle, mode_, CudnnDataType<T>::kOne(), CudnnDataType<T>::kZero(),
            data_desc_, x->template data<T>(), data_desc_,
            y->template mutable_data<T>(ctx.GetPlace()), bn_param_desc_,
            scale->template data<BatchNormParamType<T>>(),
            bias->template data<BatchNormParamType<T>>(), this_factor,
            mean_out->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace()),
            variance_out->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace()),
            epsilon, saved_mean->template mutable_data<BatchNormParamType<T>>(
                         ctx.GetPlace()),
            saved_variance->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace())));
      }
    }

    // clean when exit.
    CUDNN_ENFORCE(platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_ENFORCE(
        platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
  }
};

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void KeBNBackwardScaleBias(
    const T *dy, const T *x, const BatchNormParamType<T> *mean,
    const BatchNormParamType<T> *variance, const double epsilon, const int N,
    const int C, const int HxW, BatchNormParamType<T> *dscale,
    BatchNormParamType<T> *dbias) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ds_storage;
  __shared__ typename BlockReduce::TempStorage db_storage;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> ds_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> db_sum = static_cast<BatchNormParamType<T>>(0);

    BatchNormParamType<T> inv_var_i = 1.0 / sqrt(variance[i] + epsilon);
    BatchNormParamType<T> mean_i = mean[i];
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == framework::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      ds_sum += static_cast<BatchNormParamType<T>>(dy[index]) *
                (static_cast<BatchNormParamType<T>>(x[index]) - mean_i);
      db_sum += static_cast<BatchNormParamType<T>>(dy[index]);
    }
    ds_sum = BlockReduce(ds_storage).Reduce(ds_sum, hipcub::Sum());
    db_sum = BlockReduce(db_storage).Reduce(db_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      dscale[i] = ds_sum * inv_var_i;
      dbias[i] = db_sum;
    }
    __syncthreads();
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeBNBackwardData(const T *dy,
                                        const BatchNormParamType<T> *scale,
                                        const BatchNormParamType<T> *variance,
                                        const double epsilon, const int C,
                                        const int HxW, const int num, T *dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? i / HxW % C : i % C;
    BatchNormParamType<T> inv_var = 1.0 / sqrt(variance[c] + epsilon);
    dx[i] = static_cast<T>(static_cast<BatchNormParamType<T>>(dy[i]) *
                           scale[c] * inv_var);
  }
}

template <typename T>
class BatchNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");

    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    const auto *x = ctx.Input<Tensor>("X");
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");

    const auto &x_dims = x->dims();

    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

    // init output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    d_x->mutable_data<T>(ctx.GetPlace());
    if (d_scale && d_bias) {
      d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(scale->dims().size(), 1UL);
    PADDLE_ENFORCE_EQ(scale->dims()[0], C);

    std::vector<int> dims;
    std::vector<int> strides;
    if (data_layout == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * C * D, 1, W * D * C, D * C, C};
    }

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    if (!use_global_stats) {
      if ((N * H * W * D) == 1) {
        framework::TensorCopy(*d_y, ctx.GetPlace(), d_x);
        math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
            functor;
        functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
        functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
        return;
      }

      // ------------------- cudnn descriptors ---------------------
      hipdnnTensorDescriptor_t data_desc_;
      hipdnnTensorDescriptor_t bn_param_desc_;
      hipdnnBatchNormMode_t mode_;

      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
      if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
        LOG(ERROR) << "Provided epsilon is smaller than "
                   << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                   << "HIPDNN_BN_MIN_EPSILON instead.";
      }
      epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
#if CUDNN_VERSION_MIN(7, 0, 0)
      if (FLAGS_cudnn_batchnorm_spatial_persistent) {
        mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
      } else {
        mode_ = HIPDNN_BATCHNORM_SPATIAL;
      }
#else
      mode_ = HIPDNN_BATCHNORM_SPATIAL;
#endif

      CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_, CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
      CUDNN_ENFORCE(platform::dynload::hipdnnDeriveBNTensorDescriptor(
          bn_param_desc_, data_desc_, mode_));

      const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
      const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
      const void *saved_mean_data =
          saved_mean->template data<BatchNormParamType<T>>();
      const void *saved_var_data =
          saved_var->template data<BatchNormParamType<T>>();

      CUDNN_ENFORCE(platform::dynload::hipdnnBatchNormalizationBackward(
          dev_ctx.cudnn_handle(), mode_, CudnnDataType<T>::kOne(),
          CudnnDataType<T>::kZero(), CudnnDataType<T>::kOne(),
          CudnnDataType<T>::kZero(), data_desc_, x->template data<T>(),
          data_desc_, d_y->template data<T>(), data_desc_,
          d_x->template mutable_data<T>(ctx.GetPlace()), bn_param_desc_,
          scale->template data<BatchNormParamType<T>>(),
          d_scale->template mutable_data<BatchNormParamType<T>>(ctx.GetPlace()),
          d_bias->template mutable_data<BatchNormParamType<T>>(ctx.GetPlace()),
          epsilon, saved_mean_data, saved_var_data));

      // clean when exit.
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
    } else {
      const auto *running_mean = ctx.Input<Tensor>("Mean");
      const auto *running_var = ctx.Input<Tensor>("Variance");

      const auto *running_mean_data =
          running_mean->template data<BatchNormParamType<T>>();
      const auto *running_var_data =
          running_var->template data<BatchNormParamType<T>>();

      const int num = x->numel();
      const int block = 512;
      int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
      const int max_blocks = std::max(max_threads / block, 1);
      int grid1 = (num + block - 1) / block;
      int grid2 = std::min(C, max_blocks);

      if (data_layout == framework::DataLayout::kNCHW) {
        if (d_x) {
          KeBNBackwardData<T, framework::DataLayout::kNCHW><<<
              grid1, block, 0, dev_ctx.stream()>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<T, block, framework::DataLayout::kNCHW><<<
              grid2, block, 0, dev_ctx.stream()>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      } else {
        if (d_x) {
          KeBNBackwardData<T, framework::DataLayout::kNHWC><<<
              grid1, block, 0, dev_ctx.stream()>>>(
              d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
              running_var_data, epsilon, C, H * W, num, d_x->data<T>());
        }
        if (d_scale && d_bias) {
          KeBNBackwardScaleBias<T, block, framework::DataLayout::kNHWC><<<
              grid2, block, 0, dev_ctx.stream()>>>(
              d_y->data<T>(), x->data<T>(), running_mean_data, running_var_data,
              epsilon, N, C, H * W * D, d_scale->data<BatchNormParamType<T>>(),
              d_bias->data<BatchNormParamType<T>>());
        }
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    batch_norm, ops::BatchNormKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormKernel<plat::CUDADeviceContext, double>,
    ops::BatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    batch_norm_grad, ops::BatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, double>,
    ops::BatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
