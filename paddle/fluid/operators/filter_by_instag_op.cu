#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_cooperative_groups.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <cstring>
#include <random>
#include <string>
#include <vector>

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/timer.h"

#include "paddle/fluid/operators/filter_by_instag_op.h"

namespace cg = cooperative_groups;

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using SelectedRows = framework::SelectedRows;
using LoDTensor = framework::LoDTensor;
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
template <typename T>
using Vector = framework::Vector<T>;
#else
template <typename T>
using Vector = framework::CPUVector<T>;
#endif

#define WARP_SIZE 32
#define MAX_WARP_NUM 32

template <typename T>
__global__ void filter_copy_fuse_kernel(
    const size_t N, const int ins_per_thread, size_t* x1_lods_data,
    size_t* x2_lods_data, const int64_t* x2_data, const int64_t* x3_data,
    int64_t filter_tag_size, T* out_data, int64_t* map_data,
    size_t* map_lods_data, size_t* out_lods_data, size_t* out_idx_data,
    const T* x1_data, int x1_embed_size, int x1_lods_filled, int x2_lods_filled,
    float* loss_weight_data, int fill_value) {
  // N is instance num
  // one threads for ins_per_thread(4) instances
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int ins_start = idx * ins_per_thread;
  int ins_end = (idx + 1) * ins_per_thread;

  if (ins_start >= N) return;
  if (N < ins_end) ins_end = N;

  if (!x1_lods_filled) {
    for (int p = ins_start; p < ins_end; p++) {
      x1_lods_data[p] = p;
    }
    if (idx == 0) {
      x1_lods_data[N] = N;
    }
  }

  if (!x2_lods_filled) {
    for (int p = ins_start; p < ins_end; p++) {
      x2_lods_data[p] = p;
    }
    if (idx == 0) {
      x2_lods_data[N] = N;
    }
  }

  __syncthreads();

  // extern __shared__ int shared_data[];
  // int* flag_data = shared_data;
  // int* prefix_sum_data = (int*)(&(flag_data[N]));
  // int* mmap_aux_data = (int*)(&(prefix_sum_data[N]));

  // ================== to be optimized =============================

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int flag_data[2];
  int prefix_sum_data[2];
  int gid = idx / WARP_SIZE;

  __shared__ int shr[MAX_WARP_NUM];
  __shared__ int shr2[MAX_WARP_NUM];
  __shared__ int shr3[MAX_WARP_NUM];

  for (int p = ins_start; p < ins_end; p++) {
    int ins_tag_start = x2_lods_data[p];
    int ins_tag_end = x2_lods_data[p + 1];

    flag_data[p - ins_start] = 0;

    // filter logic
    int i = ins_tag_start;
    for (; i < ins_tag_end; i++) {
      int64_t ins_tag = x2_data[i];
      int j = 0;
      for (; j < filter_tag_size; j++) {
        if (x3_data[j] == ins_tag) break;
      }
      // if ins_tag in filter tag
      if (j < filter_tag_size) {
        flag_data[p - ins_start] = 1;
        break;
      }
    }
  }

  for (int p = ins_start; p < ins_end; p++) {
    if (p == ins_start) {
      prefix_sum_data[p - ins_start] = 0;
    } else {
      prefix_sum_data[p - ins_start] =
          flag_data[p - ins_start] * (x1_lods_data[p] - x1_lods_data[p - 1]);
    }
  }

  // prefix
  int local_addr = prefix_sum_data[ins_end - 1 - ins_start];
  int sum_addr = local_addr;

  // flag
  int local_flag = 0;
  for (int p = ins_start; p < ins_end; p++) {
    local_flag += flag_data[p - ins_start];
  }
  int sum_flag = local_flag;

  // out_lods
  int local_out_lods = 0;
  for (int p = ins_start; p < ins_end; p++) {
    local_out_lods +=
        flag_data[p - ins_start] * (x1_lods_data[p + 1] - x1_lods_data[p]);
  }
  int sum_out_lods = local_out_lods;

  // warp reduce
  for (int i = 1; i < 32; i++) {
    sum_addr += g.shfl_up(sum_addr, i);
    sum_flag += g.shfl_up(sum_flag, i);
    sum_out_lods += g.shfl_up(sum_out_lods, i);
  }

  if (g.thread_rank() == 31) {
    shr[gid] = sum_addr;
    shr2[gid] = sum_flag;
    shr3[gid] = sum_out_lods;
  }

  b.sync();

  // communicate between warp
  //
  int sum_addr2 = shr[g.thread_rank()];
  int sum_flag2 = shr2[g.thread_rank()];
  int sum_out_lods2 = shr3[g.thread_rank()];

  for (int i = 1; i < 32; i++) {
    sum_addr2 += g.shfl_up(sum_addr2, i);
    sum_flag2 += g.shfl_up(sum_flag2, i);
    sum_out_lods2 += g.shfl_up(sum_out_lods2, i);
  }

  int sum_addr3 = g.shfl(sum_addr2, gid);
  int sum_flag3 = g.shfl(sum_flag2, gid);
  int sum_out_lods3 = g.shfl(sum_out_lods2, gid);

  int p_addr = sum_addr3 - shr[gid] + sum_addr - local_addr;
  int p_flag = sum_flag3 - shr2[gid] + sum_flag - local_flag;
  int p_out_lods = sum_out_lods3 - shr3[gid] + sum_out_lods - local_out_lods;

  for (int p = ins_start; p < ins_end; p++) {
    prefix_sum_data[p - ins_start] += p_addr;
  }

  // __syncthreads() only sync threads within block
  // so we let one thread process multi idx
  // __syncthreads();
  // thread with thread id = 0 compute prefix_sum array
  // prefix_sum array is shared memory
  // extern __shared__ int prefix_sum_data[];

  // if (idx == 0) {
  //  for (int i = 0; i < N; i++) {
  //    if (i == 0) {
  //      prefix_sum_data[i] = 0;
  //    } else {
  //      prefix_sum_data[i] =
  //          prefix_sum_data[i - 1] +
  //          flag_data[i - 1] * (x1_lods_data[i] - x1_lods_data[i - 1]);
  //    }
  //  }
  // }
  // __syncthreads();

  int out_lods_idx = p_flag + 1;

  for (int p = ins_start; p < ins_end; p++) {
    if (flag_data[p - ins_start] == 1) {
      size_t batch_len = x1_lods_data[p + 1] - x1_lods_data[p];
      int t = out_lods_idx - 1;
      out_lods_data[t] = p_out_lods;
      map_data[t * 3] = (int64_t)out_lods_data[t];
      map_data[t * 3 + 1] = x1_lods_data[p];
      map_lods_data[t] = t;

      out_lods_data[out_lods_idx] =
          out_lods_data[t] + (x1_lods_data[p + 1] - x1_lods_data[p]);
      map_data[t * 3 + 2] = out_lods_data[t + 1] - out_lods_data[t];

      out_lods_idx++;
    }
  }

  /*

    if (idx == 0) {
      int out_lods_idx = 1;
      for (int i = 0; i < N; i++) {
        if (flag_data[i] == 1) {
          size_t batch_len = x1_lods_data[i + 1] - x1_lods_data[i];
          mmap_aux_data[out_lods_data[out_lods_idx - 1]] = x1_lods_data[i];
          int p = out_lods_idx - 1;
          map_data[p * 3] = (int64_t)out_lods_data[p];
          map_data[p * 3 + 1] = mmap_aux_data[out_lods_data[p]];
          map_lods_data[p] = p;
          // map_data[p * 3 + 2] = out_lods_data[p + 1] - out_lods_data[p];
          out_lods_data[out_lods_idx] =
              out_lods_data[p] + (x1_lods_data[i + 1] - x1_lods_data[i]);
          map_data[p * 3 + 2] = out_lods_data[p + 1] - out_lods_data[p];
          out_lods_idx++;
        }
      }
      *out_idx_data = out_lods_idx;
      map_lods_data[out_lods_idx - 1] = out_lods_idx - 1;
    }

  */

  // =========== to be optimized =====================================

  // fill loss_weight_data
  if (*out_idx_data > 1) {
    int out_data_num = *out_idx_data - 1;
    int out_start = ins_start;
    if (out_start < out_data_num) {
      int out_end = ins_end > out_data_num ? out_data_num : ins_end;
      for (int p = out_start; p < out_end; p++) {
        loss_weight_data[p] = fill_value;
      }
    }
  }

  // 0 1 0 1 0 1 0 1
  // 0 1 3 6 10 15 21 28 36
  // 0 0 2 2 6 6
  //__syncthreads();

  for (int p = ins_start; p < ins_end; p++) {
    // copy logic
    if (flag_data[p - ins_start] == 1) {
      auto output_start_idx = prefix_sum_data[p - ins_start];

      T* dst = out_data + output_start_idx * x1_embed_size;

      const T* src_start = x1_data + x1_lods_data[p] * x1_embed_size;
      const T* src_end = x1_data + (x1_lods_data[p + 1]) * x1_embed_size;

      // optimized
      for (const T *j = src_start; j != src_end; dst++, j++) {
        *dst = *j;
      }
    }
  }
}

/*
template <>
void filter_copy_fuse_kernel<float>(
    const size_t N, const int ins_per_thread, size_t* x1_lods_data,
    size_t* x2_lods_data, const int64_t* x2_data, const int64_t* x3_data,
    int64_t filter_tag_size, float* out_data, int64_t* map_data, size_t*
map_lods_data, size_t* out_lods_data, size_t* out_idx_data, const float*
x1_data,
    int x1_embed_size, int x1_lods_filled, int x2_lods_filled) {

  // N is instance num
  // one threads for ins_per_thread(4) instances
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int ins_start = idx * ins_per_thread;
  int ins_end = (idx + 1) * ins_per_thread;

  if (ins_start >= N) return;
  if (N < ins_end) ins_end = N;

  if (!x1_lods_filled) {
    #pragma unroll
    for (int p = ins_start; p < ins_end; p++) {
      x1_lods_data[p] = p;
    }

    if (idx == 0) {
      x1_lods_data[N] = N;
    }
  }

  if (!x2_lods_filled) {
    #pragma unroll
    for (int p = ins_start; p < ins_end; p++) {
      x2_lods_data[p] = p;
    }
    if (idx == 0) {
      x2_lods_data[N] = N;
    }
  }

  __syncthreads();

  extern __shared__ int shared_data[];
  int* flag_data = shared_data;
  int* prefix_sum_data = (int*)(&(flag_data[N]));
  int* mmap_aux_data = (int*)(&(prefix_sum_data[N]));

  #pragma unroll
  for (int p = ins_start; p < ins_end; p++) {

    int ins_tag_start = x2_lods_data[p];
    int ins_tag_end = x2_lods_data[p + 1];

    flag_data[p] = 0;
    // filter logic
    int i = ins_tag_start;
    for (; i < ins_tag_end; i++) {

      int64_t ins_tag = x2_data[i];

      int j = 0;
      for (; j < filter_tag_size; j++) {
        if (x3_data[j] == ins_tag) break;
      }

      // if ins_tag in filter tag
      if (j < filter_tag_size) {
        flag_data[p] = 1;
        break;
      }

    }

  }

  // __syncthreads() only sync threads within block
  // so we let one thread process multi idx

  __syncthreads();

  // thread with thread id = 0 compute prefix_sum array
  // prefix_sum array is shared memory
  //extern __shared__ int prefix_sum_data[];

  if (idx == 0) {

    for (int i = 0; i < N; i++) {

      if (i == 0) {
        prefix_sum_data[i] = 0;
      } else {
        prefix_sum_data[i] =
            prefix_sum_data[i - 1] +
            flag_data[i - 1] * (x1_lods_data[i] - x1_lods_data[i - 1]);
      }

    }

  }

  __syncthreads();

  if (idx == 0) {

    int out_lods_idx = 1;

    for (int i = 0; i < N; i++) {

      if (flag_data[i] == 1) {

        auto batch_len = x1_lods_data[i + 1] - x1_lods_data[i];

        mmap_aux_data[out_lods_data[out_lods_idx - 1]] = x1_lods_data[i];

        int p = out_lods_idx - 1;
        map_data[p * 3] = (int64_t)out_lods_data[p];
        map_data[p * 3 + 1] = mmap_aux_data[out_lods_data[p]];

        map_lods_data[p] = p;
        //map_data[p * 3 + 2] = out_lods_data[p + 1] - out_lods_data[p];
        out_lods_data[out_lods_idx] = out_lods_data[p] + (x1_lods_data[i + 1] -
x1_lods_data[i]);

        map_data[p * 3 + 2] = out_lods_data[p + 1] - out_lods_data[p];
        out_lods_idx++;

      }

    }

    *out_idx_data = out_lods_idx;
    map_lods_data[out_lods_idx - 1] = out_lods_idx - 1;

  }

  // 0 1 0 1 0 1 0 1
  // 0 1 3 6 10 15 21 28 36
  // 0 0 2 2 6 6
  //__syncthreads();

  #pragma unroll
  for (int p = ins_start; p < ins_end; p++) {

    // copy logic
    if (flag_data[p] == 1) {

      auto output_start_idx = prefix_sum_data[p];

      float* dst = out_data + output_start_idx * x1_embed_size;

      const float* src_start = x1_data + x1_lods_data[p] * x1_embed_size;
      const float* src_end = x1_data + (x1_lods_data[p + 1]) * x1_embed_size;

      int data_num = (x1_lods_data[p + 1] - x1_lods_data[p]) * x1_embed_size;

      float4* dst_iter = reinterpret_cast<float4*>(dst);
      const float4* src_iter = reinterpret_cast<const float4*>(src_start);
      for(int t = 0; t < data_num / 4; t++) {
        *dst_iter = *src_iter;
        dst_iter++;
        src_iter++;
      }
      // optimized
      const float* j = reinterpret_cast<const float*>(src_iter);
      float* dst_final = reinterpret_cast<float*>(dst_iter);
      for (; j != src_end; dst_final++, j++) {
        *dst_final = *j;
      }

    }

  }


}


__global__ void fill_kernel(float* data, const int data_num, const int
ins_per_thread, float fill_value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int ins_start = idx * ins_per_thread;
    int ins_end = (idx + 1) * ins_per_thread;
    if (ins_start >= data_num) return;
    if (ins_end > data_num) ins_end = data_num;
    for (int p = ins_start; p < ins_end; p++) {
      data[p] = fill_value;
    }
}

*/

template <typename T>
__global__ void copy_grad_kernel(const size_t N, const int ins_per_thread,
                                 const T* out_grad_data, T* x1_grad_data,
                                 const int64_t* map_data, int x1_embed_size) {
  // N is instance num
  // one threads for one instance
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int ins_start = idx * ins_per_thread;
  int ins_end = (idx + 1) * ins_per_thread;
  if (ins_start >= N) {
    return;
  }
  if (ins_end > N) ins_end = N;
  for (int p = ins_start; p < ins_end; p++) {
    T* dst = x1_grad_data + map_data[p * 3 + 1] * x1_embed_size;
    const T* src_start = out_grad_data + map_data[p * 3] * x1_embed_size;
    const T* src_end =
        out_grad_data + (map_data[p * 3] + map_data[p * 3 + 2]) * x1_embed_size;
    for (const T *j = src_start; j != src_end; dst++, j++) {
      *dst = *j;
    }
  }
}

/*
template <>
void copy_grad_kernel<float>(const size_t N, const int ins_per_thread, const
float* out_grad_data,
                                 float* x1_grad_data, const int64_t* map_data,
                                 int x1_embed_size) {
  // N is instance num
  // one threads for one instance
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int ins_start = idx * ins_per_thread;
  int ins_end = (idx + 1) * ins_per_thread;

  if (ins_start >= N) {
    return;
  }
  if (ins_end > N) ins_end = N;

  #pragma unroll
  for (int p = ins_start; p < ins_end; p++) {

    float* dst = x1_grad_data + map_data[p * 3 + 1] * x1_embed_size;
    const float* src_start = out_grad_data + map_data[p * 3] * x1_embed_size;
    const float* src_end =
      out_grad_data +
      (map_data[p * 3] + map_data[p * 3 + 2]) * x1_embed_size;

    int data_num = map_data[p * 3 + 2] * x1_embed_size;

    const float4* src_iter = reinterpret_cast<const float4*>(src_start);
    float4* dst_iter = reinterpret_cast<float4*>(dst);

    for( int t = 0; t < data_num / 4; t++) {
      *dst_iter = *src_iter;
      dst_iter++;
      src_iter++;
    }
    const float* j = reinterpret_cast<const float*>(src_iter);
    float* dst_final = reinterpret_cast<float*>(dst_iter);

    // using float*
    for (; j != src_end; dst_final++, j++) {
      *dst_final = *j;
    }
  }
}
*/

template <typename T>
class FilterByInstagGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    // platform::Timer timeline_;
    // timeline_.Start();

    const auto gpu_place =
        BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace());
    gpuStream_t current_stream = context.cuda_device_context().stream();
    int max_thread_num_per_block =
        context.cuda_device_context().GetMaxThreadsPerBlock();
    // X1 is global FC output
    // Dim [batch size, embedding size]
    const LoDTensor* x1 = context.Input<LoDTensor>("Ins");
    bool is_lod = context.Attr<bool>("is_lod");

    int is_x1_lod = -1;
    if (is_lod)
      is_x1_lod = 1;
    else
      is_x1_lod = 0;

    int64_t out_val_if_empty = context.Attr<int64_t>("out_val_if_empty");
    size_t x1_embed_size = x1->dims()[1];
    // X2 is ins tag list
    // LoD [[0, Sum(ins1), Sum(ins1, ins2), ... ]]
    const LoDTensor* x2 = context.Input<LoDTensor>("Ins_tag");
    // expected auto = const int64_t
    const int64_t* x2_data = x2->data<int64_t>();

    // X3 is local fc tag list
    // LoD [[0, Sum(fc1), Sum(fc1, fc2) ...]]
    const Tensor* x3 = context.Input<Tensor>("Filter_tag");
    const int64_t* x3_data = x3->data<int64_t>();

    // =========== need to be further optimized =================

    int x2_lods_filled = 0;

    Vector<size_t> x2_lods;
    // Vector, in GPU
    if (x2->lod().size() != 0) {  // lod_level = 1
      x2_lods = x2->lod()[0];
      x2_lods_filled = 1;
    } else {  // lod_level = 0
      // const size_t x2_lods_size = x2->dims()[0];
      x2_lods.resize(x2->dims()[0] + 1);
      // move to cuda
      // x2_lods.push_back(0);
      // for (size_t i = 0; i < x2_lods_size; i++) {
      //  x2_lods.push_back(i + 1);
      //}
    }

    size_t* x2_lods_data = x2_lods.CUDAMutableData(gpu_place);
    const size_t x2_lods_size = x2_lods.size() - 1;
    // Vector, in GPU
    int x1_lods_filled = 0;
    Vector<size_t> x1_lods;
    if (!is_x1_lod) {
      // move to cuda
      x1_lods.resize(x1->dims()[0] + 1);
      // x1_lods.push_back(0);
      // for (int i = 0; i < x1->dims()[0]; i++) {
      //  x1_lods.push_back(i + 1);
      //}
    } else {
      // x1_lods = context.Input<LoDTensor>("Ins")->lod()[0];
      // new: lod_level=0 => lod() return {}
      if (x1->lod().size() != 0) {  // lod_level = 1
        x1_lods_filled = 1;
        x1_lods = x1->lod()[0];

      } else {  // lod_level = 0
        x1_lods.resize(x1->dims()[0] + 1);

        // move to cuda
        // x1_lods.push_back(0);
        // for (int i = 0; i < x1->dims()[0]; i++) {
        //  x1_lods.push_back(i + 1);
        //}
      }
    }

    size_t* x1_lods_data = x1_lods.CUDAMutableData(gpu_place);
    auto* x1_data = x1->data<T>();

    // ============= need to be further optimized ===================

    // timeline_.Pause();
    // std::cout << "prephase 1 cost time: " << timeline_.ElapsedSec() <<
    // std::endl;

    // timeline_.Start();
    // set output value
    // for those whose ins been dropout, set 0 for whole lines.
    // otherwise, copy whole line
    // Dim [local fc count, batch size, embedding size]
    LoDTensor* out = context.Output<LoDTensor>("Out");
    LoDTensor* map = context.Output<LoDTensor>("IndexMap");
    LoDTensor* loss_weight = context.Output<LoDTensor>("LossWeight");

    int out_first = x1->dims()[0];
    if (x1_lods_filled) {
      out_first = x1_lods.back();
    }

    // if (x1_lods_filled) {
    //  out->Resize(framework::make_ddim(
    //    {(int64_t)x1_lods.back(), (int64_t)x1_embed_size}));
    //} else {
    out->Resize(
        framework::make_ddim({(int64_t)out_first, (int64_t)x1_embed_size}));

    //}
    map->Resize(framework::make_ddim({(int64_t)x2_lods_size, 3}));

    loss_weight->Resize(framework::make_ddim({(int64_t)x2_lods_size, 1}));

    // timeline_.Pause();
    // std::cout << "prephase 2 cost time: " << timeline_.ElapsedSec() <<
    // std::endl;

    // timeline_.Start();
    // loss_weight->Resize(
    //      framework::make_ddim({(int64_t)x2_lods_size, 1}));

    T* out_data = out->mutable_data<T>(gpu_place);
    int64_t* map_data = map->mutable_data<int64_t>(gpu_place);

    float* loss_weight_data = loss_weight->mutable_data<float>(gpu_place);

    // this is not needed if implementation is correct
    // thrust::device_ptr<T> out_data_ptr(out_data);
    // thrust::fill(out_data_ptr, out_data_ptr + out->numel(), 0);

    // timeline_.Pause();
    // std::cout << "prephase 3 cost time: " << timeline_.ElapsedSec() <<
    // std::endl;

    // timeline_.Start();
    // std::cout << "=====DEBUG====== out numel " << out->numel() << " " <<
    // x2_lods_size << " " << x1_embed_size <<std::endl;

    // not needed
    // Vector<int> flag(x2_lods_size, 0);
    // int* flag_data = flag.CUDAMutableData(context.GetPlace());

    // Vector<int> prefix_sum(x2_lods_size, 0);
    // int* prefix_sum_data = prefix_sum.CUDAMutableData(context.GetPlace());

    int block_size = max_thread_num_per_block;
    int ins_per_thread = (x2_lods_size + block_size - 1) / block_size;
    //    x2_lods_size / block_size >= 1 ? x2_lods_size / block_size + 1: 1;
    dim3 block_dim(block_size);
    dim3 grid_dim(1);

    // std::cout << "=====DEBUG====== out numel " << out->numel() << " " <<
    // x2_lods_size << " " << x1_embed_size << " " << block_size << " " <<
    // ins_per_thread <<std::endl;

    // fileter_logic
    // filter_by_instag_cuda_kernel<<<grid_dim, block_dim, 0, current_stream>>>(
    //    x2_lods_size, x2_lods_data, x2_data, x3_data, x3->numel(), flag_data);

    // filter + copy fuse
    // std::unordered_map<int64_t, int64_t> mmap_aux;

    // thrust::device_vector<int64_t> mmap_aux(x1_lods.back());

    // shared_data
    // Vector<int64_t> mmap_aux(x1_lods.back()); // space -> time

    // ============reduce time copy data from cpu to gpu================
    Vector<size_t> out_lods(x2_lods_size + 1, 0);
    Vector<size_t> map_lods(x2_lods_size + 1, 0);

    // thrust::device_vector<size_t> out_idx(1);
    Vector<size_t> out_idx(1, 0);

    // out_lods.resize(x2_lods_size + 1);
    // out_lods[0] = 0;

    size_t* out_lods_data = out_lods.CUDAMutableData(gpu_place);
    size_t* map_lods_data = map_lods.CUDAMutableData(gpu_place);

    // size_t* out_idx_data = thrust::raw_pointer_cast(&out_idx[0]);
    // int64_t* mmap_aux_data = thrust::raw_pointer_cast(&mmap_aux[0]);

    // int64_t* mmap_aux_data = mmap_aux.CUDAMutableData(context.GetPlace());
    size_t* out_idx_data = out_idx.CUDAMutableData(gpu_place);
    // ==================================================================

    // auto* out_lods_data = out_lods->mutable_data<size_t>(context.GetPlace());
    // auto* mmap_aux_data =
    // mmap_aux->mutable_data<int64_t>(context.GetPlace());
    // auto* out_idx_data = out_idx->mutable_data<size_t>(context.GetPlace());

    // timeline_.Pause();
    // std::cout << "pre phase 4 cost time: " << timeline_.ElapsedSec() <<
    // std::endl;

    // timeline_.Start();

    // filter_copy_fuse_kernel<<<grid_dim, block_dim, 0,
    //                          current_stream>>>(
    //    x2_lods_size, ins_per_thread, x1_lods_data, x2_lods_data, x2_data,
    //    x3_data, x3->numel(), flag_data, out_data, map_data, map_lods_data,
    //    out_lods_data, mmap_aux_data, out_idx_data, x1_data, x1_embed_size);

    float fill_value = 1.0;

    filter_copy_fuse_kernel<<<grid_dim, block_dim,
                              out_first * sizeof(int) +
                                  x2_lods_size*(sizeof(int) + sizeof(int)),
                              current_stream>>>(
        x2_lods_size, ins_per_thread, x1_lods_data, x2_lods_data, x2_data,
        x3_data, x3->numel(), out_data, map_data, map_lods_data, out_lods_data,
        out_idx_data, x1_data, x1_embed_size, x1_lods_filled, x2_lods_filled,
        loss_weight_data, fill_value);

    // std::cout << "============DEBUG=============flag data" << std::endl;
    // for(int i = 0; i < x2_lods_size; i++) {
    //  std::cout << flag_data[i] << " ";
    //}
    // std::cout << std::endl;

    // filter + copy fuse
    // copy_kernel<<<grid_dim_2, block_dim_2, 0, current_stream>>>(N, out_data,
    // x1_data, map_data, x1_embed_size);

    platform::GpuStreamSync(current_stream);
    // timeline_.Pause();
    // std::cout << "kernel phase cost time: " << timeline_.ElapsedSec() <<
    // std::endl;

    // timeline_.Start();
    // out_lods resize
    out_lods.resize(out_idx[0]);

    // timeline_.Pause();
    // std::cout << "kernel phase cost time: " << timeline_.ElapsedSec() << " "
    // << out_lods.size() << " " << out_idx[0] << std::endl;

    // timeline_.Start();
    // std::cout << "============DEBUG=============flag data" << std::endl;
    // thrust::device_ptr<int> flag_data_ptr(flag_data);

    // for(int i = 0; i < x2_lods_size; i++) {
    // std::cout << flag_data_ptr[i] << " ";
    //}
    // std::cout << std::endl;

    // thrust::device_ptr<const size_t> x1_lods_data_ptr(x1_lods_data);
    // std::cout << "============DEBUG=============x2_lods data" << std::endl;
    // for(int i = 0; i <= x2_lods_size; i++) {
    //  std::cout << x1_lods_data_ptr[i] << " ";
    //}
    // std::cout << std::endl;

    // std::unordered_map<int64_t, int64_t> mmap_aux;
    // Vector<size_t> out_lods;
    // out_lods.reserve(x2_lods_size + 1);
    // out_lods.push_back(0);

    // int cnt = 0;
    // for (auto it = flag.begin(); it != flag.end(); cnt++, it++) {
    //  if ((*it) == 1) {
    //    size_t batch_len = x1_lods[cnt + 1] - x1_lods[cnt];
    //    mmap_aux[out_lods.back()] = x1_lods[cnt];
    //    //out_lods.push_back(out_lods.back() + batch_len);
    //  }
    //}

    if (out_lods.size() - 1 > 0) {
      out->Resize(framework::make_ddim(
          {(int64_t)out_lods.back(), (int64_t)x1_embed_size}));

      map->Resize(framework::make_ddim({(int64_t)out_lods.size() - 1, 3}));
      loss_weight->Resize(
          framework::make_ddim({(int64_t)out_lods.size() - 1, 1}));

    } else {
      out->Resize(framework::make_ddim({1, (int64_t)x1_embed_size}));
      map->Resize(framework::make_ddim({1, 3}));
      loss_weight->Resize(framework::make_ddim({1, 1}));
    }

    // auto* out_data = out->mutable_data<T>(context.GetPlace());
    // auto* map_data = map->mutable_data<int64_t>(context.GetPlace());

    // float* loss_weight_data =
    //    loss_weight->mutable_data<float>(gpu_place);

    if (out_lods.size() - 1 > 0) {
      // move to cuda kernel
      // Vector<size_t> map_lods(out_lods.size(), 0);

      map_lods.resize(out_lods.size());
      // thrust::device_ptr<int64_t> map_data_ptr(map_data);
      // only one host -> device
      // thrust::host_vector<int64_t> h_vec(3 * (out_lods.size() - 1));

      // for (size_t i = 0; i < out_lods.size() - 1; i++) {

      // h_vec[i * 3] = (int64_t)out_lods[i];
      // h_vec[i * 3 + 1] = mmap_aux_data[(int64_t)out_lods[i]];
      // h_vec[i * 3 + 2] = out_lods[i + 1] - out_lods[i];
      //  map_lods[i] = i;

      //}

      // map_lods[out_lods.size() - 1] = out_lods.size() - 1;

      // only one copy
      // thrust::copy(h_vec.begin(), h_vec.end(), map_data_ptr);

      std::vector<Vector<size_t>> map_lod_info;
      map_lod_info.emplace_back(map_lods);
      map->set_lod(map_lod_info);
      loss_weight->set_lod(map_lod_info);
      std::vector<Vector<size_t>> out_lod_info;
      out_lod_info.emplace_back(out_lods);
      out->set_lod(out_lod_info);

      // std::cout << "=====DEBUG====== out numel " << out->numel() << " " <<
      // out_lods.back() << " " << x1_embed_size << std::endl;
      // thrust::device_ptr<T> out_data_ptr(out_data);
      // thrust::fill(out_data_ptr, out_data_ptr + out->numel(), 0);
      // can be optimized ???

      // move to cuda kernel

      // ins_per_thread = (loss_weight->numel() + block_size - 1) / block_size;
      // float fill_value = 1.0;
      // fill_kernel<<<grid_dim, block_dim, 0,
      // current_stream>>>(loss_weight_data, loss_weight->numel(),
      // ins_per_thread, fill_value);

      // platform::GpuStreamSync(current_stream);

      // thrust::device_ptr<float> loss_weight_data_ptr(loss_weight_data);
      // thrust::fill(loss_weight_data_ptr,
      //             loss_weight_data_ptr + loss_weight->numel(), 1.0);

      // only one kernel launch
      // size_t N = out_lods.size() - 1;
      // dim3 block_dim_2(block_size);
      // dim3 grid_dim_2((N + block_size - 1) / block_size);
      // copy_kernel<<<grid_dim_2, block_dim_2, 0, current_stream>>>(N,
      // out_data, x1_data, map_data, x1_embed_size);
      // hipStreamSynchronize(current_stream);

      // timeline_.Pause();
      // std::cout << "left phase cost time: " << timeline_.ElapsedSec() <<
      // std::endl;

    } else {
      Vector<size_t> map_lods(2, 0);
      thrust::device_ptr<int64_t> map_data_ptr(map_data);
      map_data_ptr[0] = 0;
      map_data_ptr[1] = 1;
      map_data_ptr[2] = 1;
      map_lods[0] = 0;
      map_lods[1] = 1;
      out_lods.push_back(1);
      std::vector<Vector<size_t>> map_lod_info;
      map_lod_info.emplace_back(map_lods);
      map->set_lod(map_lod_info);
      loss_weight->set_lod(map_lod_info);
      std::vector<Vector<size_t>> out_lod_info;
      out_lod_info.emplace_back(out_lods);
      out->set_lod(out_lod_info);
      thrust::device_ptr<T> out_data_ptr(out_data);
      // gpu kernel
      if (std::is_same<T, int32_t>::value) {
        thrust::fill(out_data_ptr, out_data_ptr + out->numel(),
                     static_cast<int32_t>(out_val_if_empty));
      } else if (std::is_same<T, int64_t>::value) {
        thrust::fill(out_data_ptr, out_data_ptr + out->numel(),
                     static_cast<int64_t>(out_val_if_empty));
      } else if (std::is_same<T, float>::value) {
        thrust::fill(out_data_ptr, out_data_ptr + out->numel(),
                     static_cast<float>(out_val_if_empty));
      } else {
        thrust::fill(out_data_ptr, out_data_ptr + out->numel(),
                     static_cast<double>(out_val_if_empty));
      }
      thrust::device_ptr<float> loss_weight_data_ptr(loss_weight_data);
      loss_weight_data_ptr[0] = 0;
    }
  }
};

template <typename T>
class FilterByInstagGradGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    // platform::Timer timeline_;
    // timeline_.Start();

    const auto gpu_place =
        BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace());
    gpuStream_t current_stream = context.cuda_device_context().stream();
    auto max_thread_num_per_block =
        context.cuda_device_context().GetMaxThreadsPerBlock();
    auto* output_grad = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* x1_grad = context.Output<LoDTensor>(framework::GradVarName("Ins"));
    auto* loss_weight = context.Input<LoDTensor>("LossWeight");
    auto* mmap = context.Input<LoDTensor>("IndexMap");
    auto* x1 = context.Input<LoDTensor>("Ins");

    x1_grad->set_lod(context.Input<LoDTensor>("Ins")->lod());
    x1_grad->Resize(x1->dims());

    auto* mmap_data = mmap->data<int64_t>();
    // expected auto = T
    auto* output_grad_data = output_grad->data<T>();
    auto* loss_weight_data = loss_weight->data<float>();

    // timeline_.Pause();

    // std::cout << "fill phase cost time: " << timeline_.ElapsedSec() <<
    // std::endl;

    // timeline_.Start();

    // expected auto = T
    auto* x1_grad_data = x1_grad->mutable_data<T>(gpu_place);
    thrust::device_ptr<T> x1_grad_data_ptr(x1_grad_data);
    thrust::device_ptr<const float> loss_weight_data_ptr(loss_weight_data);

    thrust::fill(x1_grad_data_ptr,
                 x1_grad_data_ptr + x1->dims()[0] * x1->dims()[1], 0);

    if (loss_weight->numel() != 1 || loss_weight_data_ptr[0] != 0) {
      auto output_dims = output_grad->dims();
      int x1_embed_size = output_dims[1];

      // one thread for multi-instances
      int block_size = max_thread_num_per_block;

      size_t N = mmap->dims()[0];
      dim3 block_dim(block_size);
      dim3 grid_dim((N + block_size - 1) / block_size);

      const int ins_per_thread =
          1;  // N / block_size >= 1 ? N / block_size + 1: 1;

      copy_grad_kernel<<<grid_dim, block_dim, 0, current_stream>>>(
          N, ins_per_thread, output_grad_data, x1_grad_data, mmap_data,
          x1_embed_size);

      hipStreamSynchronize(current_stream);

      // timeline_.Pause();
      // std::cout << "grad kernel phase cost time: " << timeline_.ElapsedSec()
      // << std::endl;
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(filter_by_instag, ops::FilterByInstagGPUKernel<float>,
                        ops::FilterByInstagGPUKernel<double>,
                        ops::FilterByInstagGPUKernel<int32_t>,
                        ops::FilterByInstagGPUKernel<int64_t>);

REGISTER_OP_CUDA_KERNEL(filter_by_instag_grad,
                        ops::FilterByInstagGradGPUKernel<float>,
                        ops::FilterByInstagGradGPUKernel<double>,
                        ops::FilterByInstagGradGPUKernel<int32_t>,
                        ops::FilterByInstagGradGPUKernel<int64_t>);
