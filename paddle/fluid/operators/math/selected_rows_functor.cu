#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <set>
#include <vector>

#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {
namespace math {
template <typename T>
struct SelectedRowsAdd<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::SelectedRows& input2,
                  framework::SelectedRows* output) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2.height());
    output->set_height(in1_height);

    framework::Vector<int64_t> in1_rows(input1.rows());
    auto& in2_rows = input2.rows();
    std::vector<int64_t> out_rows;
    out_rows.reserve(in1_rows.size() + in2_rows.size());

    // concat rows
    out_rows.insert(out_rows.end(), in1_rows.begin(), in1_rows.end());
    out_rows.insert(out_rows.end(), in2_rows.begin(), in2_rows.end());
    output->set_rows(out_rows);

    auto* out_value = output->mutable_value();
    auto& in1_value = input1.value();
    auto& in2_value = input2.value();

    auto in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, in2_value.numel() / in2_rows.size());
    PADDLE_ENFORCE_EQ(in1_row_numel, out_value->numel() / out_rows.size());

    auto* out_data = out_value->data<T>();
    auto* in1_data = in1_value.data<T>();

    auto in1_place = input1.place();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(in1_place), true);
    auto in2_place = input2.place();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(in2_place), true);
    auto out_place = context.GetPlace();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(out_place), true);

    memory::Copy(boost::get<platform::CUDAPlace>(out_place), out_data,
                 boost::get<platform::CUDAPlace>(in1_place), in1_data,
                 in1_value.numel() * sizeof(T), context.stream());

    auto* in2_data = in2_value.data<T>();
    memory::Copy(boost::get<platform::CUDAPlace>(out_place),
                 out_data + in1_value.numel(),
                 boost::get<platform::CUDAPlace>(in2_place), in2_data,
                 in2_value.numel() * sizeof(T), context.stream());
  }
};

template struct SelectedRowsAdd<platform::CUDADeviceContext, float>;
template struct SelectedRowsAdd<platform::CUDADeviceContext, double>;

namespace {
template <typename T, int block_size>
__global__ void SelectedRowsAddTensorKernel(const T* selected_rows,
                                            const int64_t* rows, T* tensor_out,
                                            int64_t row_numel) {
  const int ty = blockIdx.x;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we can not use
    // tensor_out[index] += selected_rows[index]; Instead, we have to use
    // AtomicAdd to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(tensor_out + index, selected_rows[index]);
  }
}
}  // namespace

template <typename T>
struct SelectedRowsAddTensor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::Tensor& input2, framework::Tensor* output) {
    auto in1_height = input1.height();
    auto in2_dims = input2.dims();
    auto out_dims = output->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);
    PADDLE_ENFORCE_EQ(in1_height, out_dims[0]);

    auto& in1_value = input1.value();
    auto& in1_rows = input1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2.numel() / in1_height);
    PADDLE_ENFORCE_EQ(in1_row_numel, output->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2.data<T>();
    auto* out_data = output->data<T>();

    SetConstant<platform::CUDADeviceContext, T> functor;
    functor(context, output, static_cast<T>(0));

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(in1_rows.size(), 1);
    SelectedRowsAddTensorKernel<
        T, block_size><<<grid, threads, 0, context.stream()>>>(
        in1_data, in1_rows.CUDAData(context.GetPlace()), out_data,
        in1_row_numel);

    auto out_eigen = framework::EigenVector<T>::Flatten(*output);
    auto in2_eigen = framework::EigenVector<T>::Flatten(input2);
    out_eigen.device(*context.eigen_device()) = out_eigen + in2_eigen;
  }
};

template struct SelectedRowsAddTensor<platform::CUDADeviceContext, float>;
template struct SelectedRowsAddTensor<platform::CUDADeviceContext, double>;
template struct SelectedRowsAdd<platform::CUDADeviceContext, platform::float16>;
template struct SelectedRowsAddTensor<platform::CUDADeviceContext,
                                      platform::float16>;

template <typename T>
struct SelectedRowsAddTo<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  const int64_t input2_offset,
                  framework::SelectedRows* input2) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2->height());

    auto& in1_rows = input1.rows();
    auto& in2_rows = *(input2->mutable_rows());

    auto& in1_value = input1.value();
    auto* in2_value = input2->mutable_value();

    // concat rows
    if (in1_rows.size()) {
      in2_rows.Extend(in1_rows.begin(), in1_rows.end());
    }

    auto in1_place = input1.place();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(in1_place), true);
    auto in2_place = input2->place();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(in2_place), true);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = in2_value->data<T>();
    memory::Copy(boost::get<platform::CUDAPlace>(in2_place),
                 in2_data + input2_offset,
                 boost::get<platform::CUDAPlace>(in1_place), in1_data,
                 in1_value.numel() * sizeof(T), context.stream());
  }
};

template struct SelectedRowsAddTo<platform::CUDADeviceContext, float>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext, double>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext, int>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext, int64_t>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext,
                                  platform::float16>;

namespace {
template <typename T, int block_size>
__global__ void SelectedRowsAddToTensorKernel(const T* selected_rows,
                                              const int64_t* rows,
                                              T* tensor_out,
                                              int64_t row_numel) {
  const int ty = blockIdx.x;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(tensor_out + index, selected_rows[index]);
  }
}
}  // namespace

template <typename T>
struct SelectedRowsAddToTensor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  framework::Tensor* input2) {
    auto in1_height = input1.height();
    auto in2_dims = input2->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);

    auto& in1_value = input1.value();
    auto& in1_rows = input1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2->data<T>();
    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(in1_rows.size(), 1);
    SelectedRowsAddToTensorKernel<
        T, block_size><<<grid, threads, 0, context.stream()>>>(
        in1_data, in1_rows.CUDAData(context.GetPlace()), in2_data,
        in1_row_numel);
  }
};

template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, float>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, double>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, int>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, int64_t>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext,
                                        platform::float16>;

namespace scatter {

template <typename T, int block_size>
__global__ void MergeAddKernel(const T* input, const int64_t* input_rows,
                               T* out, const int64_t* out_rows,
                               size_t out_rows_size, int64_t row_numel) {
  const int ty = blockIdx.x;
  int tid = threadIdx.x;
  __shared__ size_t out_idx;

  if (tid == 0) {
    for (size_t i = 0; i < out_rows_size; i++) {
      if (input_rows[ty] == out_rows[i]) {
        out_idx = i;
      }
    }
  }

  __syncthreads();

  input += ty * row_numel;
  out += out_idx * row_numel;
  for (int index = tid; index < row_numel; index += block_size) {
    paddle::platform::CudaAtomicAdd(out + index, input[index]);
  }
}

template <typename T>
struct MergeAdd<platform::CUDADeviceContext, T> {
  framework::SelectedRows operator()(const platform::CUDADeviceContext& context,
                                     const framework::SelectedRows& input,
                                     const bool sorted_result = false) {
    framework::SelectedRows out;
    (*this)(context, input, &out);
    return out;
  }

  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input,
                  framework::SelectedRows* output,
                  const bool sorted_result = false) {
    framework::Vector<int64_t> input_rows(input.rows());
    if (input_rows.size() == 0) {
      return;
    }

    framework::SelectedRows& out = *output;
    std::set<int64_t> row_set(input_rows.begin(), input_rows.end());
    std::vector<int64_t> merge_rows_cpu(row_set.begin(), row_set.end());
    framework::Vector<int64_t> merge_rows(merge_rows_cpu);

    auto input_width = input.value().dims()[1];

    out.set_rows(merge_rows);
    out.set_height(input.height());
    out.mutable_value()->mutable_data<T>(
        framework::make_ddim(
            {static_cast<int64_t>(merge_rows.size()), input_width}),
        context.GetPlace());

    math::SetConstant<platform::CUDADeviceContext, T> constant_functor;
    constant_functor(context, out.mutable_value(), static_cast<T>(0));

    auto* out_data = out.mutable_value()->data<T>();
    auto* input_data = input.value().data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid1(input_rows.size(), 1);

    MergeAddKernel<T, 256><<<grid1, threads, 0, context.stream()>>>(
        input_data, input_rows.CUDAData(context.GetPlace()), out_data,
        out.mutable_rows()->CUDAMutableData(context.GetPlace()),
        out.rows().size(), input_width);
  }

  void operator()(const platform::CUDADeviceContext& context,
                  const std::vector<const framework::SelectedRows*>& inputs,
                  framework::SelectedRows* output,
                  const bool sorted_result = false) {
    if (inputs.size() == 0) {
      VLOG(3) << "no input! return";
      return;
    }
    const framework::SelectedRows* has_value_input = nullptr;
    for (auto* in : inputs) {
      if (in->rows().size() > 0) {
        has_value_input = in;
        break;
      }
    }
    if (has_value_input == nullptr) {
      VLOG(3) << "no input has value! just return" << std::endl;
      return;
    }
    auto input_width = has_value_input->value().dims()[1];
    auto input_height = has_value_input->height();
    framework::SelectedRows& out = *output;
    std::set<int64_t> merged_row_set;
    for (auto* input : inputs) {
      if (input->rows().size() == 0) {
        continue;
      }
      PADDLE_ENFORCE_EQ(input_width, input->value().dims()[1],
                        "all input should have same "
                        "dimension except for the first one");
      PADDLE_ENFORCE_EQ(input_height, input->height(),
                        "all input should have same height");
      merged_row_set.insert(input->rows().begin(), input->rows().end());
    }
    std::vector<int64_t> merge_rows_cpu(merged_row_set.begin(),
                                        merged_row_set.end());
    framework::Vector<int64_t> merge_rows(merge_rows_cpu);

    out.set_rows(merge_rows);
    out.set_height(input_height);
    out.mutable_value()->mutable_data<T>(
        framework::make_ddim(
            {static_cast<int64_t>(merge_rows.size()), input_width}),
        context.GetPlace());

    math::SetConstant<platform::CUDADeviceContext, T> constant_functor;
    constant_functor(context, out.mutable_value(), static_cast<T>(0));

    auto* out_data = out.mutable_value()->data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);

    for (auto* input : inputs) {
      if (input->rows().size() == 0) {
        continue;
      }
      auto* input_data = input->value().data<T>();
      auto& input_rows = input->rows();
      dim3 grid1(input_rows.size(), 1);

      MergeAddKernel<T, 256><<<grid1, threads, 0, context.stream()>>>(
          input_data, input_rows.CUDAData(context.GetPlace()), out_data,
          out.mutable_rows()->CUDAMutableData(context.GetPlace()),
          out.rows().size(), input_width);
    }
  }
};

template struct MergeAdd<platform::CUDADeviceContext, float>;
template struct MergeAdd<platform::CUDADeviceContext, double>;
template struct MergeAdd<platform::CUDADeviceContext, int>;
template struct MergeAdd<platform::CUDADeviceContext, int64_t>;
template struct MergeAdd<platform::CUDADeviceContext, platform::float16>;

template <typename T, int block_size>
__global__ void UpdateToTensorKernel(const T* selected_rows,
                                     const int64_t* rows, const ScatterOps& op,
                                     T* tensor_out, int64_t row_numel) {
  const int ty = blockIdx.x;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;
  // FIXME(typhoonzero): use macro fix the below messy code.
  switch (op) {
    case ScatterOps::ASSIGN:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] = selected_rows[index];
      }
      break;
    case ScatterOps::ADD:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] += selected_rows[index];
      }
      break;
    case ScatterOps::SUB:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] -= selected_rows[index];
      }
      break;
    case ScatterOps::SUBBY:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] = selected_rows[index] - tensor_out[index];
      }
      break;
    case ScatterOps::MUL:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] *= selected_rows[index];
      }
      break;
    case ScatterOps::DIV:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] /= selected_rows[index];
      }
      break;
    case ScatterOps::DIVBY:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] = selected_rows[index] / tensor_out[index];
      }
      break;
  }
}

template <typename T>
struct UpdateToTensor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const ScatterOps& op, const framework::SelectedRows& input1,
                  framework::Tensor* input2) {
    // NOTE: Use SelectedRowsAddToTensor for better performance
    //       no additional MergeAdd called.
    MergeAdd<platform::CUDADeviceContext, T> merge_func;
    auto merged_in1 = merge_func(context, input1);

    auto in1_height = merged_in1.height();
    auto in2_dims = input2->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);

    auto& in1_value = merged_in1.value();
    auto& in1_rows = merged_in1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2->numel() / in1_height);

    auto* in1_data = in1_value.template data<T>();
    auto* in2_data = input2->data<T>();

    dim3 threads(platform::PADDLE_CUDA_NUM_THREADS, 1);
    dim3 grid(in1_rows.size(), 1);
    UpdateToTensorKernel<T, platform::PADDLE_CUDA_NUM_THREADS><<<
        grid, threads, 0, context.stream()>>>(in1_data, in1_rows.cuda_data(),
                                              op, in2_data, in1_row_numel);
  }
};
}  // namespace scatter
}  // namespace math
}  // namespace operators
}  // namespace paddle
