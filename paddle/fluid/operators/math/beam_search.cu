#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/beam_search.h"
#include "paddle/fluid/platform/cuda_device_function.h"

namespace paddle {
namespace operators {
namespace math {

struct Triple {
  __device__ __forceinline__ Triple() {}
  __device__ __forceinline__ Triple(int o, int i, float s)
      : offset(o), id(i), score(s) {}

  __device__ __forceinline__ void set(int o, int i, float s) {
    offset = o;
    id = i;
    score = s;
  }

  __device__ __forceinline__ void operator=(const Triple& in) {
    offset = in.offset;
    id = in.id;
    score = in.score;
  }

  __device__ __forceinline__ bool operator<(const float s) const {
    return score < s;
  }

  __device__ __forceinline__ bool operator<(const Triple& in) const {
    return (score < in.score) || ((score == in.score) && (offset < in.offset));
  }

  int offset;
  int id;
  float score;
};

__device__ __forceinline__ void Insert(Triple* top_beam, const Triple& p,
                                       int beam_size) {
  if (p < top_beam[beam_size - 1]) {
    return;
  }
  for (int k = beam_size - 2; k >= 0; --k) {
    if (top_beam[k] < p) {
      top_beam[k + 1] = top_beam[k];
    } else {
      top_beam[k + 1] = p;
      return;
    }
  }
  top_beam[0] = p;
}

template <int MaxThreadsPerSeq, bool IsAccumulated = true>
__device__ __forceinline__ int SelectTopBeam(
    Triple* top_beam, const int64_t* pre_ids, const float* pre_scores,
    const int64_t* ids, const float* scores, const int seq_offset_start,
    const int seq_offset_end, const int seq_width, int beam_size, int end_id,
    int used_threads) {
  // top_beam is shared memory
  const int tid = threadIdx.x;
  const int tid_of_seq = threadIdx.x % MaxThreadsPerSeq;

  int num_used_threads = used_threads;

  Triple* top_beam_local = top_beam + tid * beam_size;
  if (tid_of_seq < num_used_threads) {
    for (int i = 0; i < beam_size; ++i) {
      top_beam_local[i].set(-1, -1, -INFINITY);
    }

    for (int offset = seq_offset_start; offset < seq_offset_end; ++offset) {
      int pre_id = static_cast<int>(pre_ids[offset]);
      if (pre_id == end_id) {
        if (tid_of_seq == 0) {
          Triple tmp(offset, end_id, pre_scores[offset]);
          Insert(top_beam_local, tmp, beam_size);
        }
      } else {
        int index = offset * seq_width + tid_of_seq;
        if (!IsAccumulated) {
          float pre_score = pre_scores[offset];
          for (int i = tid_of_seq; i < seq_width; i += num_used_threads) {
            float score = pre_score + __logf(scores[index]);
            int id = ids ? static_cast<int>(ids[index]) : i;
            Triple tmp(offset, id, score);
            Insert(top_beam_local, tmp, beam_size);
            index += num_used_threads;
          }
        } else {
          for (int i = tid_of_seq; i < seq_width; i += num_used_threads) {
            int id = ids ? static_cast<int>(ids[index]) : i;
            float score = scores[index];
            Triple tmp(offset, id, score);
            Insert(top_beam_local, tmp, beam_size);
            index += num_used_threads;
          }
        }
      }
    }
  }

  while (num_used_threads > 1) {
    if (num_used_threads > 16) {
      __syncthreads();
    }

    if ((num_used_threads & 0x1) != 0) {
      // If num_used_threads is a odd number, merge local top_beam of thread 0
      // and num_used_threads - 1
      if (tid_of_seq == 0) {
        int index_in_sh = (num_used_threads - 1 + tid) * beam_size;
        for (int i = 0; i < beam_size; i++) {
          Insert(top_beam_local, top_beam[index_in_sh], beam_size);
          index_in_sh++;
        }
      }
    }

    num_used_threads = num_used_threads >> 1;
    if (tid_of_seq < num_used_threads) {
      int index_in_sh = (num_used_threads + tid) * beam_size;
      for (int i = 0; i < beam_size; i++) {
        Insert(top_beam_local, top_beam[index_in_sh], beam_size);
        index_in_sh++;
      }
    }
  }

  if (tid_of_seq == 0) {
    int num_items = 0;
    for (int i = 0; i < beam_size; ++i) {
      num_items =
          (top_beam_local[i].score > -INFINITY) ? num_items + 1 : num_items;
    }
    return num_items;
  }

  return 0;
}

__device__ __forceinline__ bool PruneEndBeams(Triple* top_beam_local,
                                              const int64_t* pre_ids,
                                              const int end_id, int num_items) {
  bool finish_flag = true;
  for (int i = 0; i < num_items; ++i) {
    int offset = top_beam_local[i].offset;
    if (top_beam_local[i].id != end_id ||
        static_cast<int>(pre_ids[offset]) != end_id) {
      finish_flag = false;
      break;
    }
  }
  return finish_flag;
}

template <bool ReturnParentIdx = false>
__device__ __forceinline__ void WriteBack(
    int64_t* selected_ids, float* selected_scores, int* parent_idx,
    size_t* selected_offsets, Triple* top_beam_local,
    const int seq_offset_start, const int seq_offset_end,
    const int selected_seq_start, const int selected_seq_length) {
  const int tid = threadIdx.x;  // use 1 thread only for each sequence
  int global_index = selected_seq_start;
  for (int global_offset = seq_offset_start; global_offset < seq_offset_end;
       ++global_offset) {
    for (int local_index = 0; local_index < selected_seq_length;
         ++local_index) {
      if (top_beam_local[local_index].offset == global_offset) {
        selected_ids[global_index] =
            static_cast<int64_t>(top_beam_local[local_index].id);
        selected_scores[global_index] = top_beam_local[local_index].score;
        if (ReturnParentIdx) {
          parent_idx[global_index] = static_cast<int>(global_offset);
        }
        global_index++;
      }
    }
    selected_offsets[global_offset + 1] = static_cast<size_t>(global_index);
  }
}

template <int MaxLength, int MaxThreadsPerSeq, int MaxSeqs>
__device__ void BeamSearchDetails(
    int64_t* selected_ids, float* selected_scores, int* parent_idx,
    size_t* selected_offsets, const int64_t* pre_ids, const float* pre_scores,
    const int64_t* ids, const float* scores, const int seq_offset_start,
    const int seq_offset_end, const int seq_width, int beam_size, int end_id,
    bool is_accumulated, int num_used_threads) {
  __shared__ Triple top_beam[MaxLength];

  int num_items = 0;
  if (is_accumulated) {
    num_items = SelectTopBeam<MaxThreadsPerSeq, true>(
        top_beam, pre_ids, pre_scores, ids, scores, seq_offset_start,
        seq_offset_end, seq_width, beam_size, end_id, num_used_threads);
  } else {
    num_items = SelectTopBeam<MaxThreadsPerSeq, false>(
        top_beam, pre_ids, pre_scores, ids, scores, seq_offset_start,
        seq_offset_end, seq_width, beam_size, end_id, num_used_threads);
  }

  const int tid = threadIdx.x;  // use 1 thread only for each sequence
  const int tid_of_seq = tid % MaxThreadsPerSeq;
  if (tid_of_seq == 0) {
    // Use 1 thread for each sequence.
    Triple* top_beam_local = top_beam + tid * beam_size;
    bool finish_flag =
        PruneEndBeams(top_beam_local, pre_ids, end_id, num_items);

    int selected_seq_start = 0;
    int selected_seq_length = finish_flag ? 0 : num_items;

    if (MaxSeqs > 1) {
      const int seq_id = (MaxSeqs > 1) ? tid / MaxThreadsPerSeq : tid;
      __shared__ int shared_mem[MaxSeqs];

      // [0, MaxSeqs - 1], length of each sequences
      shared_mem[seq_id] = selected_seq_length;
      __syncthreads();

      for (int s = 0; s < seq_id; ++s) {
        selected_seq_start += shared_mem[s];
      }

      if (seq_id == 0) {
        selected_offsets[0] = 0;
      }
    } else {
      selected_offsets[0] = 0;
    }

    if (parent_idx) {
      WriteBack<true>(selected_ids, selected_scores, parent_idx,
                      selected_offsets, top_beam_local, seq_offset_start,
                      seq_offset_end, selected_seq_start, selected_seq_length);
    } else {
      WriteBack<false>(selected_ids, selected_scores, parent_idx,
                       selected_offsets, top_beam_local, seq_offset_start,
                       seq_offset_end, selected_seq_start, selected_seq_length);
    }
  }
}

template <int MaxLength, int MaxThreadsPerSeq, int MaxSeqs>
__global__ void BeamSearchKernel(int64_t* selected_ids, float* selected_scores,
                                 int* parent_idx, size_t* selected_offsets,
                                 const int64_t* pre_ids,
                                 const float* pre_scores, const int64_t* ids,
                                 const float* scores, const size_t* seq_offsets,
                                 const int num_seqs, const int seq_width,
                                 int beam_size, int end_id, bool is_accumulated,
                                 int num_used_threads) {
  const int tid = threadIdx.x;
  const int seq_id = (MaxSeqs > 1) ? tid / MaxThreadsPerSeq : tid;

  int seq_offset_start = static_cast<int>(seq_offsets[seq_id]);
  int seq_offset_end = static_cast<int>(seq_offsets[seq_id + 1]);

  BeamSearchDetails<MaxLength, MaxThreadsPerSeq, MaxSeqs>(
      selected_ids, selected_scores, parent_idx, selected_offsets, pre_ids,
      pre_scores, ids, scores, seq_offset_start, seq_offset_end, seq_width,
      beam_size, end_id, is_accumulated, num_used_threads);
}

template <int MaxLength, int MaxThreadsPerSeq>
__global__ void BeamSearchKernelSingle(
    int64_t* selected_ids, float* selected_scores, int* parent_idx,
    size_t* selected_offsets, const int64_t* pre_ids, const float* pre_scores,
    const int64_t* ids, const float* scores, const int seq_length,
    const int seq_width, int beam_size, int end_id, bool is_accumulated,
    int num_used_threads) {
  const int seq_offset_start = 0;
  const int seq_offset_end = seq_length;

  BeamSearchDetails<MaxLength, MaxThreadsPerSeq, 1>(
      selected_ids, selected_scores, parent_idx, selected_offsets, pre_ids,
      pre_scores, ids, scores, seq_offset_start, seq_offset_end, seq_width,
      beam_size, end_id, is_accumulated, num_used_threads);
}

static inline int GetNumUsedThreads(const int max_threads_per_seq,
                                    const int seq_width, int beam_size) {
  int num_used_threads = (seq_width + beam_size - 1) / beam_size;
  num_used_threads = max_threads_per_seq < num_used_threads
                         ? max_threads_per_seq
                         : num_used_threads;

  num_used_threads =
      num_used_threads > 32
          ? (num_used_threads >> 5) << 5
          : (num_used_threads > 16
                 ? 32
                 : (num_used_threads > 8
                        ? 16
                        : (num_used_threads > 4
                               ? 8
                               : (num_used_threads > 2 ? 4
                                                       : num_used_threads))));
  return num_used_threads;
}

template <typename T>
class BeamSearchFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::LoDTensor* pre_ids,
                  const framework::LoDTensor* pre_scores,
                  const framework::LoDTensor* ids,
                  const framework::LoDTensor* scores,
                  framework::LoDTensor* selected_ids,
                  framework::LoDTensor* selected_scores,
                  framework::Tensor* parent_idx, size_t level, size_t beam_size,
                  int end_id, bool is_accumulated) {
    auto abs_lod = framework::ToAbsOffset(scores->lod());

    const int64_t* pre_ids_data = pre_ids->data<int64_t>();
    const float* pre_scores_data = pre_scores->data<float>();
    const int64_t* ids_data = ids ? ids->data<int64_t>() : nullptr;
    const float* scores_data = scores->data<float>();

    const size_t num_seqs = abs_lod[level].size() - 1;
    size_t seq_width = 1;
    for (int i = 1; i < scores->dims().size(); i++) {
      seq_width *= scores->dims()[i];
    }

    // Reserve a big enough memory.
    auto selected_dims =
        framework::make_ddim({static_cast<int64_t>(num_seqs * beam_size), 1});
    int64_t* selected_ids_data =
        selected_ids->mutable_data<int64_t>(selected_dims, context.GetPlace());
    float* selected_scores_data =
        selected_scores->mutable_data<float>(selected_dims, context.GetPlace());
    int* parent_idx_data =
        parent_idx
            ? parent_idx->mutable_data<int>(
                  {static_cast<int64_t>(num_seqs * beam_size)},
                  context.GetPlace())
            : nullptr;

    framework::LoD selected_lod(2);
    selected_lod[0].assign(abs_lod[level].begin(), abs_lod[level].end());
    selected_lod[1].resize(scores->dims()[0] + 1);
    size_t* selected_offsets =
        selected_lod[1].CUDAMutableData(context.GetPlace());

    if (num_seqs == 1) {
      const int seq_length = static_cast<int>(abs_lod[level][1]);
      const int kMaxThreadsPerSeq = 1024;
      int num_used_threads =
          GetNumUsedThreads(kMaxThreadsPerSeq, static_cast<int>(seq_width),
                            static_cast<int>(beam_size));
      switch (platform::RoundToPowerOfTwo(beam_size * seq_width)) {
        CUDA_LAUNCH_KERNEL_HELPER(
            BeamSearchKernelSingle<kPowerOfTwoDim, kMaxThreadsPerSeq><<<
                1, kMaxThreadsPerSeq, 0, context.stream()>>>(
                selected_ids_data, selected_scores_data, parent_idx_data,
                selected_offsets, pre_ids_data, pre_scores_data, ids_data,
                scores_data, seq_length, static_cast<int>(seq_width),
                static_cast<int>(beam_size), static_cast<int>(end_id),
                is_accumulated, num_used_threads));
      }
    } else if (num_seqs <= 4) {
      const size_t* seq_offsets = abs_lod[level].CUDAData(context.GetPlace());
      // Use only 1 block
      const int kMaxThreadsPerSeq = 32;
      const int kMaxSeqs = 4;
      int num_used_threads =
          GetNumUsedThreads(kMaxThreadsPerSeq, static_cast<int>(seq_width),
                            static_cast<int>(beam_size));
      switch (platform::RoundToPowerOfTwo(beam_size * num_seqs * 32)) {
        CUDA_LAUNCH_KERNEL_HELPER(
            BeamSearchKernel<kPowerOfTwoDim, kMaxThreadsPerSeq, kMaxSeqs><<<
                1, num_seqs * kMaxThreadsPerSeq, 0, context.stream()>>>(
                selected_ids_data, selected_scores_data, parent_idx_data,
                selected_offsets, pre_ids_data, pre_scores_data, ids_data,
                scores_data, seq_offsets, static_cast<int>(num_seqs),
                static_cast<int>(seq_width), static_cast<int>(beam_size),
                end_id, is_accumulated, num_used_threads));
      }
    } else {
      LOG(FATAL) << "Not implemented.";
    }

    context.Wait();
    if (!framework::CheckLoD(selected_lod)) {
      PADDLE_THROW("lod %s is not right", framework::LoDToString(selected_lod));
    }

    selected_ids->set_lod(selected_lod);
    selected_scores->set_lod(selected_lod);
    if (selected_lod[1].back() < num_seqs * beam_size) {
      auto final_selected_dims = framework::make_ddim(
          {static_cast<int64_t>(selected_lod[1].back()), 1});
      selected_ids->Resize(final_selected_dims);
      selected_scores->Resize(final_selected_dims);
      if (parent_idx) {
        parent_idx->Resize({static_cast<int64_t>(selected_lod[1].back())});
      }
    }
  }
};

template class BeamSearchFunctor<platform::CUDADeviceContext, int>;
template class BeamSearchFunctor<platform::CUDADeviceContext, int64_t>;
template class BeamSearchFunctor<platform::CUDADeviceContext, float>;
template class BeamSearchFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
