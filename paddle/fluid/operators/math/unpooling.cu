#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/unpooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {
template <typename T>
__global__ void KernelUnpool2dMax(const int nthreads, const T* input_data,
                                  const int* indices_data,
                                  const int input_height, const int input_width,
                                  const int channels, T* output_data,
                                  const int output_height,
                                  const int output_width) {
  int in_n_stride = input_height * input_width * channels;
  int in_c_stride = input_height * input_width;
  int out_n_stride = output_height * output_width * channels;
  int out_c_stride = output_height * output_width;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int bidx = i / in_n_stride;
    int boffset = i % in_n_stride;
    int cidx = boffset / in_c_stride;
    int out_offset = bidx * out_n_stride + cidx * out_c_stride;
    int out_index = indices_data[i];
    PADDLE_ASSERT(out_index < out_c_stride);
    output_data[out_offset + out_index] = input_data[i];
  }
}
template <typename T>
__global__ void KernelUnpool2dMaxGrad(
    const int nthreads, const T* input_data, const int* indices_data,
    const int input_height, const int input_width, const int channels,
    const T* output_data, const T* output_grad, const int output_height,
    const int output_width, T* input_grad) {
  int in_n_stride = input_height * input_width * channels;
  int in_c_stride = input_height * input_width;
  int out_n_stride = output_height * output_width * channels;
  int out_c_stride = output_height * output_width;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int bidx = i / in_n_stride;
    int boffset = i % in_n_stride;
    int cidx = boffset / in_c_stride;
    int out_offset = bidx * out_n_stride + cidx * out_c_stride;
    int out_index = indices_data[i];
    PADDLE_ASSERT(out_index < out_c_stride);
    input_grad[i] = output_grad[out_offset + out_index];
  }
}
/*
 * All tensors are in NCHW format.
 */
template <typename T>
class Unpool2dMaxFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& indices, framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const T* input_data = input.data<T>();
    const int* indices_data = indices.data<int>();
    T* output_data = output->mutable_data<T>(context.GetPlace());
    int threads = 1024;
    int grid = (input.numel() + threads - 1) / threads;
    KernelUnpool2dMax<T><<<grid, threads, 0, context.stream()>>>(
        input.numel(), input_data, indices_data, input_height, input_width,
        output_channels, output_data, output_height, output_width);
  }
};
/*
 * All tensors are in NCHW format.
 */
template <typename T>
class Unpool2dMaxGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& indices,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const T* input_data = input.data<T>();
    const int* indices_data = indices.data<int>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());
    int threads = 1024;
    int grid = (input.numel() + threads - 1) / threads;
    KernelUnpool2dMaxGrad<T><<<grid, threads, 0, context.stream()>>>(
        input.numel(), input_data, indices_data, input_height, input_width,
        output_channels, output_data, output_grad_data, output_height,
        output_width, input_grad_data);
  }
};
template class Unpool2dMaxGradFunctor<platform::CUDADeviceContext, float>;
template class Unpool2dMaxGradFunctor<platform::CUDADeviceContext, double>;
template class Unpool2dMaxFunctor<platform::CUDADeviceContext, float>;
template class Unpool2dMaxFunctor<platform::CUDADeviceContext, double>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
