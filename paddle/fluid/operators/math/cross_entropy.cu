#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/operators/math.h"
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T, typename LabelT>
__global__ void CrossEntropyKernel(T* Y, const T* X, const LabelT* label,
                                   const int N, const int D,
                                   const int ignore_index) {
  CUDA_KERNEL_LOOP(i, N) {
    auto lbl = static_cast<int64_t>(label[i]);
    PADDLE_ENFORCE(lbl >= 0 && lbl < D || lbl == ignore_index,
                   "The value of label[%d] expected >= 0 and < %ld, or == %ld, "
                   "but got %ld. Please check input value.",
                   i, D, ignore_index, lbl);
    Y[i] = ignore_index == lbl
               ? static_cast<T>(0)
               : -math::TolerableValue<T>()(real_log(X[i * D + lbl]));
  }
}

template <typename T>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int class_num) {
  int tid = threadIdx.x;
  T val(0);

  int idx = blockIdx.x * class_num + tid;
  int end = blockIdx.x * class_num + class_num;
  for (; idx < end; idx += blockDim.x) {
    val += math::TolerableValue<T>()(real_log(X[idx])) * label[idx];
  }

  val = paddle::platform::reduceSum(val, tid, blockDim.x);
  if (threadIdx.x == 0) {
    Y[blockIdx.x] = -val;
  }
}

template <typename T>
struct HardLabelCrossEntropyCUDAFunctorImpl {
 public:
  HardLabelCrossEntropyCUDAFunctorImpl(T* loss_data, const T* prob_data,
                                       const void* label_data,
                                       const int batch_size,
                                       const int class_num,
                                       const int ignore_index,
                                       const int block_size, gpuStream_t stream)
      : loss_data_(loss_data),
        prob_data_(prob_data),
        label_data_(label_data),
        batch_size_(batch_size),
        class_num_(class_num),
        ignore_index_(ignore_index),
        block_size_(block_size),
        stream_(stream) {}

  template <typename U>
  void apply() const {
    int grid_size = (batch_size_ + block_size_ - 1) / block_size_;
    CrossEntropyKernel<T, U><<<grid_size, block_size_, 0, stream_>>>(
        loss_data_, prob_data_, static_cast<const U*>(label_data_), batch_size_,
        class_num_, ignore_index_);
  }

 private:
  T* loss_data_;
  const T* prob_data_;
  const void* label_data_;
  const int batch_size_;
  const int class_num_;
  const int ignore_index_;
  const int block_size_;
  gpuStream_t stream_;
};

template <typename T>
class CrossEntropyFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& ctx,
                  framework::Tensor* out, const framework::Tensor* prob,
                  const framework::Tensor* labels, const bool softLabel,
                  const int ignore_index, const int axis_dim) {
    const T* prob_data = prob->data<T>();
    T* loss_data = out->mutable_data<T>(ctx.GetPlace());

    int batch_size = prob->dims()[0];
    int class_num = prob->dims()[1];
#ifdef __HIPCC__
    constexpr int kMaxBlockDim = 256;
#else
    constexpr int kMaxBlockDim = 512;
#endif

    if (softLabel) {
      const T* label_data = labels->data<T>();
      int block = class_num > kMaxBlockDim
                      ? kMaxBlockDim
                      : pow(2, static_cast<int>(std::log2(class_num)));

      SoftCrossEntropyKernel<T><<<batch_size, block, 0, ctx.stream()>>>(
          loss_data, prob_data, label_data, class_num);
    } else {
      HardLabelCrossEntropyCUDAFunctorImpl<T> functor(
          loss_data, prob_data, labels->data(), batch_size, class_num,
          ignore_index, kMaxBlockDim, ctx.stream());
      framework::VisitDataType(framework::TransToProtoVarType(labels->dtype()),
                               functor);
    }
  }
};

template class CrossEntropyFunctor<platform::CUDADeviceContext, float>;
template class CrossEntropyFunctor<platform::CUDADeviceContext, double>;
template class CrossEntropyFunctor<platform::CUDADeviceContext,
                                   platform::float16>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
