#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math.h"
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__global__ void CrossEntropyKernel(T* Y, const T* X, const int64_t* label,
                                   const int N, const int D,
                                   const int ignore_index) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    PADDLE_ENFORCE(label[i] >= 0 && label[i] < D || label[i] == ignore_index,
                   "label[%d] expected >= 0 and < %ld, or == %ld, but got "
                   "%ld. Please check input value.",
                   i, D, ignore_index, label[i]);
    Y[i] = ignore_index == label[i]
               ? static_cast<T>(0)
               : -math::TolerableValue<T>()(real_log(X[i * D + label[i]]));
  }
}

template <typename T>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int class_num) {
  int tid = threadIdx.x;
  T val(0);

  int idx = blockIdx.x * class_num + tid;
  int end = blockIdx.x * class_num + class_num;
  for (; idx < end; idx += blockDim.x) {
    val += math::TolerableValue<T>()(real_log(X[idx])) * label[idx];
  }

  val = paddle::platform::reduceSum(val, tid, blockDim.x);
  if (threadIdx.x == 0) {
    Y[blockIdx.x] = -val;
  }
}

template <typename T>
class CrossEntropyFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& ctx,
                  framework::Tensor* out, const framework::Tensor* prob,
                  const framework::Tensor* labels, const bool softLabel,
                  const int ignore_index, const int axis_dim) {
    const T* prob_data = prob->data<T>();
    T* loss_data = out->mutable_data<T>(ctx.GetPlace());

    int batch_size = prob->dims()[0];
    int class_num = prob->dims()[1];

    if (softLabel) {
      const T* label_data = labels->data<T>();
      int block = class_num > 512
                      ? 512
                      : pow(2, static_cast<int>(std::log2(class_num)));

      SoftCrossEntropyKernel<T><<<batch_size, block, 0, ctx.stream()>>>(
          loss_data, prob_data, label_data, class_num);
    } else {
      const int64_t* label_data = labels->data<int64_t>();
      int block = 512;
      int grid = (batch_size + block - 1) / block;
      CrossEntropyKernel<T><<<grid, block, 0, ctx.stream()>>>(
          loss_data, prob_data, label_data, batch_size, class_num,
          ignore_index);
    }
  }
};

template class CrossEntropyFunctor<platform::CUDADeviceContext, float>;
template class CrossEntropyFunctor<platform::CUDADeviceContext, double>;
template class CrossEntropyFunctor<platform::CUDADeviceContext,
                                   platform::float16>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
