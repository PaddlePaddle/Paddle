#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "paddle/fluid/operators/math/depthwise_conv.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__inline__ __device__ T warpReduceSum(T val) {
#if CUDA_VERSION < 9000
  for (int offset = 16; offset > 0; offset /= 2)
    val += __shfl_down(val, offset);
  return val;
#else
#define FULL_MASK 0xffffffff
  for (int offset = 16; offset > 0; offset /= 2)
    val += __shfl_down_sync(FULL_MASK, val, offset);
  return val;
#endif
}
__forceinline__ __device__ unsigned lane_id() {
  unsigned ret;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(ret));
  return ret;
}

__forceinline__ __device__ unsigned warp_id() {
  unsigned ret;
  asm volatile("mov.u32 %0, %warpid;" : "=r"(ret));
  return ret;
}

#define ARG_DEFINE_KernelDepthwiseConv                                         \
  const T *const input_data, const T *const filter_data, const int batch_size, \
      const int output_channels, const int output_height,                      \
      const int output_width, const int input_channels,                        \
      const int input_height, const int input_width,                           \
      const int filter_multiplier, const int filter_height,                    \
      const int filter_width, const int stride_height, const int stride_width, \
      const int padding_height, const int padding_width,                       \
      const int dilate_height, const int dilate_width, T *const output_data

// A Cuda kernel to compute the depthwise convolution forward pass
// in NCHW format.
template <typename T>
__device__ __inline__ void KernelDepthwiseConv(ARG_DEFINE_KernelDepthwiseConv) {
  for (int w_out = threadIdx.x; w_out < output_width; w_out += blockDim.x) {
    for (int h_out = threadIdx.y; h_out < output_height; h_out += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_out = blockIdx.x;

      const int c_in = c_out / filter_multiplier;
      const T* weight = filter_data + c_out * filter_height * filter_width;
      T value = 0;
      const int h_in_start = -padding_height + h_out * stride_height;
      const int w_in_start = -padding_width + w_out * stride_width;
      const int h_in_end = h_in_start + filter_height * dilate_height;
      const int w_in_end = w_in_start + filter_width * dilate_width;

      const int in_offset =
          ((batch * input_channels + c_in) * input_height) * input_width;

      const int h_end = h_in_end < input_height ? h_in_end : input_height;
      const int w_end = w_in_end < input_width ? w_in_end : input_width;
      const int h_start = h_in_start > 0 ? h_in_start : 0;
      const int w_start = w_in_start > 0 ? w_in_start : 0;
      int weight_offset = 0;

      for (int h_in = h_in_start; h_in < h_in_end; h_in += dilate_height) {
        for (int w_in = w_in_start; w_in < w_in_end; w_in += dilate_width) {
          if (h_in >= h_start && h_in < h_end && w_in >= w_start &&
              w_in < w_end) {
            const int offset = in_offset + h_in * input_width + w_in;
            value += weight[weight_offset] * input_data[offset];
          }
          weight_offset++;
        }
      }
      int index =
          ((batch * gridDim.x + c_out) * output_height + h_out) * output_width +
          w_out;
      output_data[index] = value;
    }
  }
}

template <typename T, int c_filter>
__device__ __inline__ void KernelDepthwiseConvCFilter(
    ARG_DEFINE_KernelDepthwiseConv) {
  const int kWeghtSize = c_filter * c_filter;
  T r_weight[kWeghtSize];
  const int batch = blockIdx.y;
  const int c_out = blockIdx.x;
  const T* weight = filter_data + c_out * c_filter * c_filter;
  for (int i = 0; i < c_filter * c_filter; i++) r_weight[i] = weight[i];

  for (int w_out = threadIdx.x; w_out < output_width; w_out += blockDim.x) {
    for (int h_out = threadIdx.y; h_out < output_height; h_out += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_out = blockIdx.x;

      const int c_in = c_out / filter_multiplier;
      T value = 0;
      const int h_in_start = -padding_height + h_out * stride_height;
      const int w_in_start = -padding_width + w_out * stride_width;
      const int h_in_end = h_in_start + c_filter * dilate_height;
      const int w_in_end = w_in_start + c_filter * dilate_width;

      const int in_offset =
          ((batch * input_channels + c_in) * input_height) * input_width;

      const int h_end = h_in_end < input_height ? h_in_end : input_height;
      const int w_end = w_in_end < input_width ? w_in_end : input_width;
      const int h_start = h_in_start > 0 ? h_in_start : 0;
      const int w_start = w_in_start > 0 ? w_in_start : 0;

      for (int h_in = h_in_start, h_f = 0; h_f < c_filter;
           h_in += dilate_height, h_f++) {
        for (int w_in = w_in_start, w_f = 0; w_f < c_filter;
             w_in += dilate_width, w_f++) {
          if (h_in >= 0 && h_in < input_height && w_in >= 0 &&
              w_in < input_width) {
            const int offset = in_offset + h_in * input_width + w_in;
            value += r_weight[h_f * c_filter + w_f] * input_data[offset];
          }
        }
      }
      int index =
          ((batch * gridDim.x + c_out) * output_height + h_out) * output_width +
          w_out;
      output_data[index] = value;
    }
  }
}

template <typename T, int c_filter_multiplier, int c_stride, int c_filter>
__global__ void KernelDepthwiseConvSp(ARG_DEFINE_KernelDepthwiseConv) {
  if (c_filter_multiplier == 0) {
    if (c_filter == -1)
      KernelDepthwiseConv<T>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          filter_multiplier, filter_height, filter_width, stride_height,
          stride_width, padding_height, padding_width, dilate_height,
          dilate_width, output_data);
    else
      KernelDepthwiseConvCFilter<T, c_filter>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          filter_multiplier, filter_height, filter_width, stride_height,
          stride_width, padding_height, padding_width, dilate_height,
          dilate_width, output_data);
  } else {
    if (c_filter == -1)
      KernelDepthwiseConv<T>(input_data, filter_data, batch_size,
                             output_channels, output_height, output_width,
                             input_channels, input_height, input_width,
                             c_filter_multiplier, filter_height, filter_height,
                             c_stride, c_stride, padding_height, padding_width,
                             dilate_height, dilate_width, output_data);
    else
      KernelDepthwiseConvCFilter<T, c_filter>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          c_filter_multiplier, filter_height, filter_height, c_stride, c_stride,
          padding_height, padding_width, dilate_height, dilate_width,
          output_data);
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t input.
#define ARG_DEFINE_KernelDepthwiseConvInputGrad                                \
  const T *const output_grad_data, const T *const filter_data,                 \
      const int batch_size, const int output_channels,                         \
      const int output_height, const int output_width,                         \
      const int input_channels, const int input_height, const int input_width, \
      const int filter_multiplier, const int filter_height,                    \
      const int filter_width, const int stride_height, const int stride_width, \
      const int padding_height, const int padding_width,                       \
      const int dilate_height, const int dilate_width,                         \
      T *const input_grad_data

template <typename T>
__device__ __inline__ void KernelDepthwiseConvInputGrad(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  for (int w_in = threadIdx.x; w_in < input_width; w_in += blockDim.x) {
    for (int h_in = threadIdx.y; h_in < input_height; h_in += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_in = blockIdx.x;

      const int c_out_start = c_in * filter_multiplier;

      int h_out_start =
          h_in - (filter_height - 1) * dilate_height + padding_height;

      int h_out_end = h_in + padding_height;

      int w_out_start =
          w_in - (filter_width - 1) * dilate_width + padding_width;

      int w_out_end = w_in + padding_width;

      T value = 0;

      for (int c_out = c_out_start; c_out < c_out_start + filter_multiplier;
           c_out++) {
        int filter_offset = (c_out + 1) * filter_height * filter_width;
        for (int h_out = h_out_start; h_out <= h_out_end;
             h_out += dilate_height) {
          for (int w_out = w_out_start; w_out <= w_out_end;
               w_out += dilate_width) {
            filter_offset--;
            int s_h_out = h_out / stride_height;
            int s_w_out = w_out / stride_width;
            if (h_out % stride_height == 0 && w_out % stride_width == 0 &&
                s_h_out >= 0 && s_h_out < output_height && s_w_out >= 0 &&
                s_w_out < output_width) {
              const int output_grad_offset =
                  ((batch * output_channels + c_out) * output_height +
                   s_h_out) *
                      output_width +
                  s_w_out;
              value += output_grad_data[output_grad_offset] *
                       filter_data[filter_offset];
            }
          }
        }
      }
      int index =
          ((batch * gridDim.x + c_in) * input_height + h_in) * input_width +
          w_in;
      input_grad_data[index] = value;
    }
  }
}

template <typename T, int c_filter, int c_filter_multiplier>
__device__ __inline__ void KernelDepthwiseConvInputGradCFilter(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  const int kWeghtSize = c_filter * c_filter * c_filter_multiplier + 1;
  T r_weight[kWeghtSize];
  const int batch = blockIdx.y;
  const int c_in = blockIdx.x;

  for (int c_i = 0; c_i < filter_multiplier; c_i++) {
    int c_out = c_in * filter_multiplier + c_i;
    const T* weight = filter_data + c_out * c_filter * c_filter;
    for (int i = 0; i < c_filter * c_filter; i++)
      r_weight[i + c_i * c_filter * c_filter] =
          weight[c_filter * c_filter - i - 1];
  }

  for (int w_in = threadIdx.x; w_in < input_width; w_in += blockDim.x) {
    for (int h_in = threadIdx.y; h_in < input_height; h_in += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_in = blockIdx.x;

      int h_out_start = h_in - (c_filter - 1) * dilate_height + padding_height;

      int w_out_start = w_in - (c_filter - 1) * dilate_width + padding_width;

      T value = 0;

      for (int c_i = 0; c_i < filter_multiplier; c_i++) {
        int c_out = c_in * filter_multiplier + c_i;
        for (int h_out = h_out_start, h_f = 0; h_f < c_filter;
             h_out += dilate_height, h_f++) {
          for (int w_out = w_out_start, w_f = 0; w_f < c_filter;
               w_out += dilate_width, w_f++) {
            int s_h_out = h_out / stride_height;
            int s_w_out = w_out / stride_width;
            if (h_out % stride_height == 0 && w_out % stride_width == 0 &&
                s_h_out >= 0 && s_h_out < output_height && s_w_out >= 0 &&
                s_w_out < output_width) {
              const int output_grad_offset =
                  ((batch * output_channels + c_out) * output_height +
                   s_h_out) *
                      output_width +
                  s_w_out;
              value +=
                  output_grad_data[output_grad_offset] *
                  r_weight[h_f * c_filter + w_f + c_i * c_filter * c_filter];
            }
          }
        }
      }
      int index =
          ((batch * gridDim.x + c_in) * input_height + h_in) * input_width +
          w_in;
      input_grad_data[index] = value;
    }
  }
}

template <typename T, int c_filter_multiplier, int c_stride, int c_filter>
__global__ void KernelDepthwiseConvInputGradSp(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  if (c_filter_multiplier == 0)
    KernelDepthwiseConvInputGrad<T>(
        output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, input_grad_data);
  else if (c_filter == -1)
    KernelDepthwiseConvInputGrad<T>(
        output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, c_stride, c_stride,
        padding_height, padding_width, dilate_height, dilate_width,
        input_grad_data);
  else
    KernelDepthwiseConvInputGradCFilter<T, c_filter, c_filter_multiplier>(
        output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, c_stride, c_stride,
        padding_height, padding_width, dilate_height, dilate_width,
        input_grad_data);
}

// Cuda kernel to compute the depthwise convolution backprop w.r.t. filter.
template <typename T>
__device__ __inline__ void KernelDepthwiseConvFilterGrad(
    const T* output_grad_data, const T* input_data, const int num,
    const int output_channels, const int output_height, const int output_width,
    const int input_channels, const int input_height, const int input_width,
    const int filter_multiplier, const int filter_height,
    const int filter_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, const int dilate_height,
    const int dilate_width, T* filter_grad_data) {
  T s = 0;

  int gbid = ((blockIdx.z * gridDim.y) + blockIdx.y) * gridDim.x + blockIdx.x;
  int lid = lane_id();

  for (int image_w = threadIdx.x; image_w < output_width;
       image_w += blockDim.x) {
    for (int bid = 0; bid < num; bid++) {
      for (int image_h = threadIdx.y; image_h < output_height;
           image_h += blockDim.y) {
        int kernel_id = blockIdx.z;
        int kernel_h = blockIdx.y * dilate_height - padding_height;
        int kernel_w = blockIdx.x * dilate_width - padding_width;

        int image_hk = image_h * stride_height + kernel_h;
        int image_wk = image_w * stride_width + kernel_w;
        if (image_hk < 0 || image_hk >= input_height) continue;
        if (image_wk < 0 || image_wk >= input_width) continue;
#define gaid(N, C, H, W) \
  ((((N)*gridDim.z + (C)) * output_height + (H)) * output_width + (W))

        s += output_grad_data[gaid(bid, kernel_id, image_h, image_w)] *
             input_data[((bid * (gridDim.z / filter_multiplier) +
                          kernel_id / filter_multiplier) *
                             input_height +
                         image_hk) *
                            input_width +
                        image_wk];

#undef gaid
      }
    }
  }
#if __CUDA_ARCH__ >= 530
  s = warpReduceSum<T>(s);
  if (lid == 0) paddle::platform::CudaAtomicAdd(&filter_grad_data[gbid], s);
#else
  paddle::platform::CudaAtomicAdd(&filter_grad_data[gbid], s);
#endif
}

template <typename T, int c_filter_multiplier>
__global__ void KernelDepthwiseConvFilterGradSp(
    const T* output_grad_data, const T* input_data, const int num,
    const int output_channels, const int output_height, const int output_width,
    const int input_channels, const int input_height, const int input_width,
    const int filter_multiplier, const int filter_height,
    const int filter_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, const int dilate_height,
    const int dilate_width, T* filter_grad_data) {
  if (c_filter_multiplier == 0)
    KernelDepthwiseConvFilterGrad<T>(
        output_grad_data, input_data, num, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, filter_grad_data);
  else
    KernelDepthwiseConvFilterGrad<T>(
        output_grad_data, input_data, num, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, filter_grad_data);
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <class T>
class DepthwiseConvFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& filter,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = filter.dims()[2];
    const int ksize_width = filter.dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* filter_data = filter.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int thread = 512;
    int blocks = std::min(std::max(thread / output_width, 1), output_height);
    dim3 threads(std::min(output_width, thread), blocks, 1);
    dim3 grid(output_channels, batch_size, 1);
    int filter_multiplier = output_channels / input_channels;
#define check_case(c_filter_multiplier, c_stride, c_filter)                  \
  if (c_filter_multiplier == 0 ||                                            \
      filter_multiplier == c_filter_multiplier &&                            \
          stride_height == stride_width && stride_height == c_stride &&      \
          (ksize_height == ksize_width && ksize_height == c_filter ||        \
           c_filter == -1)) {                                                \
    KernelDepthwiseConvSp<T, c_filter_multiplier, c_stride,                  \
                          c_filter><<<grid, threads, 0, context.stream()>>>( \
        input_data, filter_data, batch_size, output_channels, output_height, \
        output_width, input_channels, input_height, input_width,             \
        filter_multiplier, ksize_height, ksize_width, stride_height,         \
        stride_width, padding_height, padding_width, dilate_height,          \
        dilate_width, output_data);                                          \
    return;                                                                  \
  }
    check_case(1, 1, 3);
    check_case(1, 1, 5);
    check_case(1, 1, -1);
    check_case(1, 2, 3);
    check_case(1, 2, 5);
    check_case(1, 2, -1);
    check_case(0, 0, 3);
    check_case(0, 0, 5);
    check_case(0, 0, -1);
// NOTE(liangdun): 0,0 for other case
// add other case if needed, e.g. check_case(2^n,1)
#undef check_case
  }
};

template <typename T>
class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& filter,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output_grad.dims()[1];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = filter.dims()[2];
    const int ksize_width = filter.dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* filter_data = filter.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int thread = 512;
    int blocks = std::min(std::max(thread / input_width, 1), input_height);
    dim3 threads(std::min(input_width, thread), blocks, 1);
    dim3 grid(input_channels, batch_size, 1);
    int filter_multiplier = output_channels / input_channels;

#define check_case(c_filter_multiplier, c_stride, c_filter)             \
  if (c_filter_multiplier == 0 ||                                       \
      filter_multiplier == c_filter_multiplier &&                       \
          stride_height == stride_width && stride_height == c_stride && \
          (ksize_height == ksize_width && ksize_height == c_filter ||   \
           c_filter == -1)) {                                           \
    KernelDepthwiseConvInputGradSp<                                     \
        T, c_filter_multiplier, c_stride,                               \
        c_filter><<<grid, threads, 0, context.stream()>>>(              \
        output_grad_data, filter_data, batch_size, output_channels,     \
        output_height, output_width, input_channels, input_height,      \
        input_width, filter_multiplier, ksize_height, ksize_width,      \
        stride_height, stride_width, padding_height, padding_width,     \
        dilate_height, dilate_width, input_grad_data);                  \
    return;                                                             \
  }
    check_case(1, 1, 3);
    check_case(1, 1, 5);
    check_case(1, 1, -1);
    check_case(1, 2, 3);
    check_case(1, 2, 5);
    check_case(1, 2, -1);
    check_case(2, 1, 3);
    check_case(2, 1, 5);
    check_case(2, 1, -1);
    check_case(2, 2, 3);
    check_case(2, 2, 5);
    check_case(2, 2, -1);
    check_case(0, 0, -1);
// NOTE(liangdun): 0,0 for other case
// add other case if needed, e.g. check_case(2^n,1)
#undef check_case
  }
};

template <typename T>
class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* filter_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output_grad.dims()[1];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = filter_grad->dims()[2];
    const int ksize_width = filter_grad->dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* filter_grad_data = filter_grad->mutable_data<T>(context.GetPlace());

    int block_size = 512;
    int crop_output_height =
        std::min(std::max(block_size / output_width, 1), output_height);
    dim3 grid(ksize_width, ksize_height, output_channels);
    dim3 threads(std::min(output_width, block_size), crop_output_height, 1);
    int filter_multiplier = output_channels / input_channels;

#define check_case(c_filter_multiplier)                                       \
  if (c_filter_multiplier == 0 || c_filter_multiplier == filter_multiplier) { \
    KernelDepthwiseConvFilterGradSp<                                          \
        T, c_filter_multiplier><<<grid, threads, 0, context.stream()>>>(      \
        output_grad_data, input_data, batch_size, output_channels,            \
        output_height, output_width, input_channels, input_height,            \
        input_width, filter_multiplier, ksize_height, ksize_width,            \
        stride_height, stride_width, padding_height, padding_width,           \
        dilate_height, dilate_width, filter_grad_data);                       \
    return;                                                                   \
  }
    check_case(1);
    check_case(0);
#undef check_case
  }
};

template class DepthwiseConvFunctor<platform::CUDADeviceContext, float>;
template class DepthwiseConvFunctor<platform::CUDADeviceContext, double>;

template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext,
                                             float>;
template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext,
                                             double>;

template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              float>;
template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
