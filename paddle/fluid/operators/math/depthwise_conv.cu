#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/operators/math/depthwise_conv.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__device__ __inline__ void CudaAtomicAddWithWarp(T* sum, T value) {
  typedef hipcub::WarpReduce<T> WarpReduce;
  typename WarpReduce::TempStorage temp_storage;
  value = WarpReduce(temp_storage).Sum(value);
  if (hipcub::LaneId() == 0) platform::CudaAtomicAdd(sum, value);
}

#define ARG_DEFINE_KernelDepthwiseConv                                         \
  const T *const input_data, const T *const filter_data, const int batch_size, \
      const int output_channels, const int output_height,                      \
      const int output_width, const int input_channels,                        \
      const int input_height, const int input_width,                           \
      const int filter_multiplier, const int filter_height,                    \
      const int filter_width, const int stride_height, const int stride_width, \
      const int padding_height, const int padding_width,                       \
      const int dilate_height, const int dilate_width, T *const output_data

// A Cuda kernel to compute the depthwise convolution forward pass
// in NCHW format.
template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConv(ARG_DEFINE_KernelDepthwiseConv) {
  for (int w_out = threadIdx.x; w_out < output_width; w_out += blockDim.x) {
    for (int h_out = threadIdx.y; h_out < output_height; h_out += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_out = blockIdx.x;

      const int c_in = c_out / filter_multiplier;
      const T* weight = filter_data + c_out * filter_height * filter_width;
      T value = 0;
      const int h_in_start = -padding_height + h_out * stride_height;
      const int w_in_start = -padding_width + w_out * stride_width;
      const int h_in_end = h_in_start + filter_height * dilate_height;
      const int w_in_end = w_in_start + filter_width * dilate_width;

      const int in_offset =
          ((batch * input_channels + c_in) * input_height) * input_width;

      const int h_end = h_in_end < input_height ? h_in_end : input_height;
      const int w_end = w_in_end < input_width ? w_in_end : input_width;
      const int h_start = h_in_start > 0 ? h_in_start : 0;
      const int w_start = w_in_start > 0 ? w_in_start : 0;
      int weight_offset = 0;

      for (int h_in = h_in_start; h_in < h_in_end; h_in += dilate_height) {
        for (int w_in = w_in_start; w_in < w_in_end; w_in += dilate_width) {
          if (h_in >= h_start && h_in < h_end && w_in >= w_start &&
              w_in < w_end) {
            const int offset = in_offset + h_in * input_width + w_in;
            if (fuse_relu_before_conv) {
              value += weight[weight_offset] * max(0.0f, input_data[offset]);
            } else {
              value += weight[weight_offset] * input_data[offset];
            }
          }
          weight_offset++;
        }
      }
      int index =
          ((batch * gridDim.x + c_out) * output_height + h_out) * output_width +
          w_out;
      output_data[index] = value;
    }
  }
}

template <typename T, int c_filter, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvCFilter(
    ARG_DEFINE_KernelDepthwiseConv) {
  const int kWeghtSize = c_filter * c_filter;
  T r_weight[kWeghtSize];
  const int batch = blockIdx.y;
  const int c_out = blockIdx.x;
  const T* weight = filter_data + c_out * c_filter * c_filter;
  for (int i = 0; i < c_filter * c_filter; i++) r_weight[i] = weight[i];

  for (int w_out = threadIdx.x; w_out < output_width; w_out += blockDim.x) {
    for (int h_out = threadIdx.y; h_out < output_height; h_out += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_out = blockIdx.x;

      const int c_in = c_out / filter_multiplier;
      T value = 0;
      const int h_in_start = -padding_height + h_out * stride_height;
      const int w_in_start = -padding_width + w_out * stride_width;
      const int h_in_end = h_in_start + c_filter * dilate_height;
      const int w_in_end = w_in_start + c_filter * dilate_width;

      const int in_offset =
          ((batch * input_channels + c_in) * input_height) * input_width;

      const int h_end = h_in_end < input_height ? h_in_end : input_height;
      const int w_end = w_in_end < input_width ? w_in_end : input_width;
      const int h_start = h_in_start > 0 ? h_in_start : 0;
      const int w_start = w_in_start > 0 ? w_in_start : 0;

      for (int h_in = h_in_start, h_f = 0; h_f < c_filter;
           h_in += dilate_height, h_f++) {
        for (int w_in = w_in_start, w_f = 0; w_f < c_filter;
             w_in += dilate_width, w_f++) {
          if (h_in >= 0 && h_in < input_height && w_in >= 0 &&
              w_in < input_width) {
            const int offset = in_offset + h_in * input_width + w_in;
            if (fuse_relu_before_conv) {
              value += r_weight[h_f * c_filter + w_f] *
                       max(0.0f, input_data[offset]);
            } else {
              value += r_weight[h_f * c_filter + w_f] * input_data[offset];
            }
          }
        }
      }
      int index =
          ((batch * gridDim.x + c_out) * output_height + h_out) * output_width +
          w_out;
      output_data[index] = value;
    }
  }
}

template <typename T, int c_filter_multiplier, int c_stride, int c_filter,
          bool fuse_relu_before_conv>
__global__ void KernelDepthwiseConvSp(ARG_DEFINE_KernelDepthwiseConv) {
  if (c_filter_multiplier == 0) {
    if (c_filter == -1)
      KernelDepthwiseConv<T, fuse_relu_before_conv>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          filter_multiplier, filter_height, filter_width, stride_height,
          stride_width, padding_height, padding_width, dilate_height,
          dilate_width, output_data);
    else
      KernelDepthwiseConvCFilter<T, c_filter, fuse_relu_before_conv>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          filter_multiplier, filter_height, filter_width, stride_height,
          stride_width, padding_height, padding_width, dilate_height,
          dilate_width, output_data);
  } else {
    if (c_filter == -1)
      KernelDepthwiseConv<T, fuse_relu_before_conv>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          c_filter_multiplier, filter_height, filter_height, c_stride, c_stride,
          padding_height, padding_width, dilate_height, dilate_width,
          output_data);
    else
      KernelDepthwiseConvCFilter<T, c_filter, fuse_relu_before_conv>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          c_filter_multiplier, filter_height, filter_height, c_stride, c_stride,
          padding_height, padding_width, dilate_height, dilate_width,
          output_data);
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t input.
#define ARG_DEFINE_KernelDepthwiseConvInputGrad                                \
  const T *const input_data, const T *const output_grad_data,                  \
      const T *const filter_data, const int batch_size,                        \
      const int output_channels, const int output_height,                      \
      const int output_width, const int input_channels,                        \
      const int input_height, const int input_width,                           \
      const int filter_multiplier, const int filter_height,                    \
      const int filter_width, const int stride_height, const int stride_width, \
      const int padding_height, const int padding_width,                       \
      const int dilate_height, const int dilate_width,                         \
      T *const input_grad_data

template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvInputGrad(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  for (int w_in = threadIdx.x; w_in < input_width; w_in += blockDim.x) {
    for (int h_in = threadIdx.y; h_in < input_height; h_in += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_in = blockIdx.x;

      const int c_out_start = c_in * filter_multiplier;

      int h_out_start =
          h_in - (filter_height - 1) * dilate_height + padding_height;

      int h_out_end = h_in + padding_height;

      int w_out_start =
          w_in - (filter_width - 1) * dilate_width + padding_width;

      int w_out_end = w_in + padding_width;

      T value = 0;
      int index =
          ((batch * gridDim.x + c_in) * input_height + h_in) * input_width +
          w_in;
      if (fuse_relu_before_conv) {
        if (input_data[index] <= 0) {
          input_grad_data[index] = 0;
          continue;
        }
      }

      for (int c_out = c_out_start; c_out < c_out_start + filter_multiplier;
           c_out++) {
        int filter_offset = (c_out + 1) * filter_height * filter_width;
        for (int h_out = h_out_start; h_out <= h_out_end;
             h_out += dilate_height) {
          for (int w_out = w_out_start; w_out <= w_out_end;
               w_out += dilate_width) {
            filter_offset--;
            int s_h_out = h_out / stride_height;
            int s_w_out = w_out / stride_width;
            if (h_out % stride_height == 0 && w_out % stride_width == 0 &&
                s_h_out >= 0 && s_h_out < output_height && s_w_out >= 0 &&
                s_w_out < output_width) {
              const int output_grad_offset =
                  ((batch * output_channels + c_out) * output_height +
                   s_h_out) *
                      output_width +
                  s_w_out;
              value += output_grad_data[output_grad_offset] *
                       filter_data[filter_offset];
            }
          }
        }
      }
      input_grad_data[index] = value;
    }
  }
}

template <typename T, int c_filter, int c_filter_multiplier,
          bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvInputGradCFilter(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  const int kWeghtSize = c_filter * c_filter * c_filter_multiplier + 1;
  T r_weight[kWeghtSize];
  const int batch = blockIdx.y;
  const int c_in = blockIdx.x;

  for (int c_i = 0; c_i < filter_multiplier; c_i++) {
    int c_out = c_in * filter_multiplier + c_i;
    const T* weight = filter_data + c_out * c_filter * c_filter;
    for (int i = 0; i < c_filter * c_filter; i++)
      r_weight[i + c_i * c_filter * c_filter] =
          weight[c_filter * c_filter - i - 1];
  }

  for (int w_in = threadIdx.x; w_in < input_width; w_in += blockDim.x) {
    for (int h_in = threadIdx.y; h_in < input_height; h_in += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_in = blockIdx.x;

      int h_out_start = h_in - (c_filter - 1) * dilate_height + padding_height;

      int w_out_start = w_in - (c_filter - 1) * dilate_width + padding_width;

      T value = 0;
      int index =
          ((batch * gridDim.x + c_in) * input_height + h_in) * input_width +
          w_in;
      if (fuse_relu_before_conv) {
        if (input_data[index] <= 0) {
          input_grad_data[index] = 0;
          continue;
        }
      }

      for (int c_i = 0; c_i < filter_multiplier; c_i++) {
        int c_out = c_in * filter_multiplier + c_i;
        for (int h_out = h_out_start, h_f = 0; h_f < c_filter;
             h_out += dilate_height, h_f++) {
          for (int w_out = w_out_start, w_f = 0; w_f < c_filter;
               w_out += dilate_width, w_f++) {
            int s_h_out = h_out / stride_height;
            int s_w_out = w_out / stride_width;
            if (h_out % stride_height == 0 && w_out % stride_width == 0 &&
                s_h_out >= 0 && s_h_out < output_height && s_w_out >= 0 &&
                s_w_out < output_width) {
              const int output_grad_offset =
                  ((batch * output_channels + c_out) * output_height +
                   s_h_out) *
                      output_width +
                  s_w_out;
              value +=
                  output_grad_data[output_grad_offset] *
                  r_weight[h_f * c_filter + w_f + c_i * c_filter * c_filter];
            }
          }
        }
      }
      input_grad_data[index] = value;
    }
  }
}

template <typename T, int c_filter_multiplier, int c_stride, int c_filter,
          bool fuse_relu_before_conv>
__global__ void KernelDepthwiseConvInputGradSp(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  if (c_filter_multiplier == 0)
    KernelDepthwiseConvInputGrad<T, fuse_relu_before_conv>(
        input_data, output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, input_grad_data);
  else if (c_filter == -1)
    KernelDepthwiseConvInputGrad<T, fuse_relu_before_conv>(
        input_data, output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, c_stride, c_stride,
        padding_height, padding_width, dilate_height, dilate_width,
        input_grad_data);
  else
    KernelDepthwiseConvInputGradCFilter<T, c_filter, c_filter_multiplier,
                                        fuse_relu_before_conv>(
        input_data, output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, c_stride, c_stride,
        padding_height, padding_width, dilate_height, dilate_width,
        input_grad_data);
}

// Cuda kernel to compute the depthwise convolution backprop w.r.t. filter.
template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvFilterGrad(
    const T* output_grad_data, const T* input_data, const int num,
    const int output_channels, const int output_height, const int output_width,
    const int input_channels, const int input_height, const int input_width,
    const int filter_multiplier, const int filter_height,
    const int filter_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, const int dilate_height,
    const int dilate_width, T* filter_grad_data) {
  T s = 0;

  int gbid = ((blockIdx.z * gridDim.y) + blockIdx.y) * gridDim.x + blockIdx.x;

  for (int image_w = threadIdx.x; image_w < output_width;
       image_w += blockDim.x) {
    for (int bid = 0; bid < num; bid++) {
      for (int image_h = threadIdx.y; image_h < output_height;
           image_h += blockDim.y) {
        int kernel_id = blockIdx.z;
        int kernel_h = blockIdx.y * dilate_height - padding_height;
        int kernel_w = blockIdx.x * dilate_width - padding_width;

        int image_hk = image_h * stride_height + kernel_h;
        int image_wk = image_w * stride_width + kernel_w;
        if (image_hk < 0 || image_hk >= input_height) continue;
        if (image_wk < 0 || image_wk >= input_width) continue;
#define gaid(N, C, H, W) \
  ((((N)*gridDim.z + (C)) * output_height + (H)) * output_width + (W))
        int input_id = ((bid * (gridDim.z / filter_multiplier) +
                         kernel_id / filter_multiplier) *
                            input_height +
                        image_hk) *
                           input_width +
                       image_wk;
        if (fuse_relu_before_conv) {
          s += output_grad_data[gaid(bid, kernel_id, image_h, image_w)] *
               max(0.0f, input_data[input_id]);
        } else {
          s += output_grad_data[gaid(bid, kernel_id, image_h, image_w)] *
               input_data[input_id];
        }

#undef gaid
      }
    }
  }
  CudaAtomicAddWithWarp(&filter_grad_data[gbid], s);
}

template <typename T, int c_filter_multiplier, bool fuse_relu_before_conv>
__global__ void KernelDepthwiseConvFilterGradSp(
    const T* output_grad_data, const T* input_data, const int num,
    const int output_channels, const int output_height, const int output_width,
    const int input_channels, const int input_height, const int input_width,
    const int filter_multiplier, const int filter_height,
    const int filter_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, const int dilate_height,
    const int dilate_width, T* filter_grad_data) {
  if (c_filter_multiplier == 0)
    KernelDepthwiseConvFilterGrad<T, fuse_relu_before_conv>(
        output_grad_data, input_data, num, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, filter_grad_data);
  else
    KernelDepthwiseConvFilterGrad<T, fuse_relu_before_conv>(
        output_grad_data, input_data, num, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, filter_grad_data);
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <class T, bool fuse_relu_before_conv>
class DepthwiseConvFunctor<platform::CUDADeviceContext, T,
                           fuse_relu_before_conv> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& filter,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = filter.dims()[2];
    const int ksize_width = filter.dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* filter_data = filter.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int thread = 512;
    if (output_width > 1024 && output_width <= 2048)
      thread = (output_width - 1) / 2 + 1;
    else if (output_width > 512 && output_width <= 1024)
      thread = output_width;
    int blocks = std::min(std::max(thread / output_width, 1), output_height);
    dim3 threads(std::min(output_width, thread), blocks, 1);
    dim3 grid(output_channels, batch_size, 1);
    int filter_multiplier = output_channels / input_channels;
#define check_case(c_filter_multiplier, c_stride, c_filter)                  \
  if (c_filter_multiplier == 0 ||                                            \
      filter_multiplier == c_filter_multiplier &&                            \
          stride_height == stride_width && stride_height == c_stride &&      \
          (ksize_height == ksize_width && ksize_height == c_filter ||        \
           c_filter == -1)) {                                                \
    KernelDepthwiseConvSp<                                                   \
        T, c_filter_multiplier, c_stride, c_filter,                          \
        fuse_relu_before_conv><<<grid, threads, 0, context.stream()>>>(      \
        input_data, filter_data, batch_size, output_channels, output_height, \
        output_width, input_channels, input_height, input_width,             \
        filter_multiplier, ksize_height, ksize_width, stride_height,         \
        stride_width, padding_height, padding_width, dilate_height,          \
        dilate_width, output_data);                                          \
    return;                                                                  \
  }
    check_case(1, 1, 3);
    check_case(1, 1, 5);
    check_case(1, 1, -1);
    check_case(1, 2, 3);
    check_case(1, 2, 5);
    check_case(1, 2, -1);
    check_case(2, 1, 3);
    check_case(2, 1, 5);
    check_case(2, 1, -1);
    check_case(2, 2, 3);
    check_case(2, 2, 5);
    check_case(2, 2, -1);
    check_case(0, 0, -1);
// NOTE(liangdun): 0,0 for other case
// add other case if needed, e.g. check_case(2^n,1)
#undef check_case
  }
};

template <typename T, bool fuse_relu_before_conv>
class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, T,
                                    fuse_relu_before_conv> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& filter,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output_grad.dims()[1];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = filter.dims()[2];
    const int ksize_width = filter.dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* filter_data = filter.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int thread = 512;
    if (input_width > 1024 && input_width <= 2048)
      thread = (input_width - 1) / 2 + 1;
    else if (input_width > 512 && input_width <= 1024)
      thread = input_width;
    int blocks = std::min(std::max(thread / input_width, 1), input_height);
    dim3 threads(std::min(input_width, thread), blocks, 1);
    dim3 grid(input_channels, batch_size, 1);
    int filter_multiplier = output_channels / input_channels;

#define check_case(c_filter_multiplier, c_stride, c_filter)             \
  if (c_filter_multiplier == 0 ||                                       \
      filter_multiplier == c_filter_multiplier &&                       \
          stride_height == stride_width && stride_height == c_stride && \
          (ksize_height == ksize_width && ksize_height == c_filter ||   \
           c_filter == -1)) {                                           \
    KernelDepthwiseConvInputGradSp<                                     \
        T, c_filter_multiplier, c_stride, c_filter,                     \
        fuse_relu_before_conv><<<grid, threads, 0, context.stream()>>>( \
        input_data, output_grad_data, filter_data, batch_size,          \
        output_channels, output_height, output_width, input_channels,   \
        input_height, input_width, filter_multiplier, ksize_height,     \
        ksize_width, stride_height, stride_width, padding_height,       \
        padding_width, dilate_height, dilate_width, input_grad_data);   \
    return;                                                             \
  }
    check_case(1, 1, 3);
    check_case(1, 1, 5);
    check_case(1, 1, -1);
    check_case(1, 2, 3);
    check_case(1, 2, 5);
    check_case(1, 2, -1);
    check_case(2, 1, 3);
    check_case(2, 1, 5);
    check_case(2, 1, -1);
    check_case(2, 2, 3);
    check_case(2, 2, 5);
    check_case(2, 2, -1);
    check_case(0, 0, -1);
// NOTE(liangdun): 0,0 for other case
// add other case if needed, e.g. check_case(2^n,1)
#undef check_case
  }
};

template <typename T, bool fuse_relu_before_conv>
class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext, T,
                                     fuse_relu_before_conv> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* filter_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output_grad.dims()[1];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = filter_grad->dims()[2];
    const int ksize_width = filter_grad->dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* filter_grad_data = filter_grad->mutable_data<T>(context.GetPlace());

    int block_size = 512;
    if (output_width > 1024 && output_width <= 2048)
      block_size = (output_width - 1) / 2 + 1;
    else if (output_width > 512 && output_width <= 1024)
      block_size = output_width;
    int crop_output_height =
        std::min(std::max(block_size / output_width, 1), output_height);
    dim3 grid(ksize_width, ksize_height, output_channels);
    dim3 threads(std::min(output_width, block_size), crop_output_height, 1);
    int filter_multiplier = output_channels / input_channels;

#define check_case(c_filter_multiplier)                                       \
  if (c_filter_multiplier == 0 || c_filter_multiplier == filter_multiplier) { \
    KernelDepthwiseConvFilterGradSp<                                          \
        T, c_filter_multiplier,                                               \
        fuse_relu_before_conv><<<grid, threads, 0, context.stream()>>>(       \
        output_grad_data, input_data, batch_size, output_channels,            \
        output_height, output_width, input_channels, input_height,            \
        input_width, filter_multiplier, ksize_height, ksize_width,            \
        stride_height, stride_width, padding_height, padding_width,           \
        dilate_height, dilate_width, filter_grad_data);                       \
    return;                                                                   \
  }
    check_case(1);
    check_case(0);
#undef check_case
  }
};

template class DepthwiseConvFunctor<platform::CUDADeviceContext, float, false>;
template class DepthwiseConvFunctor<platform::CUDADeviceContext, double, false>;

template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, float,
                                             false>;
template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext,
                                             double, false>;

template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              float, false>;
template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              double, false>;

template class DepthwiseConvFunctor<platform::CUDADeviceContext, float, true>;
template class DepthwiseConvFunctor<platform::CUDADeviceContext, double, true>;

template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, float,
                                             true>;
template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext,
                                             double, true>;

template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              float, true>;
template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              double, true>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
