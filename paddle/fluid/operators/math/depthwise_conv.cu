#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/math/depthwise_conv.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__device__ __inline__ void CudaAtomicAddWithWarp(T* sum, T value) {
  typedef hipcub::WarpReduce<T> WarpReduce;
  typename WarpReduce::TempStorage temp_storage;

#ifdef __HIPCC__
  int block_size = min(blockDim.x * blockDim.y * blockDim.z, warpSize);
  value = WarpReduce(temp_storage).Sum(value, block_size);
#else
  value = WarpReduce(temp_storage).Sum(value);
#endif

  if (hipcub::LaneId() == 0) platform::CudaAtomicAdd(sum, value);
}

#define ARG_DEFINE_KernelDepthwiseConv                                         \
  const T *const input_data, const T *const filter_data, const int batch_size, \
      const int output_channels, const int output_height,                      \
      const int output_width, const int input_channels,                        \
      const int input_height, const int input_width,                           \
      const int filter_multiplier, const int filter_height,                    \
      const int filter_width, const int stride_height, const int stride_width, \
      const int padding_height, const int padding_width,                       \
      const int dilate_height, const int dilate_width, T *const output_data,   \
      const DataLayout data_layout = DataLayout::kNCHW

// A Cuda kernel to compute the depthwise convolution forward pass
// in NCHW format.
template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvNCHW(
    ARG_DEFINE_KernelDepthwiseConv) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= (output_channels * batch_size * output_height * output_width))
    return;

  const int w_out = idx % output_width;
  const int h_out = (idx / output_width) % output_height;
  const int c_out = (idx / output_width / output_height) % output_channels;
  const int batch = idx / output_width / output_height / output_channels;

  const int c_in = c_out / filter_multiplier;
  const T* weight = filter_data + c_out * filter_height * filter_width;
  T value = 0;
  const int h_in_start = -padding_height + h_out * stride_height;
  const int w_in_start = -padding_width + w_out * stride_width;
  const int h_in_end = h_in_start + filter_height * dilate_height;
  const int w_in_end = w_in_start + filter_width * dilate_width;

  int in_offset =
      ((batch * input_channels + c_in) * input_height) * input_width;

  const int h_end = h_in_end < input_height ? h_in_end : input_height;
  const int w_end = w_in_end < input_width ? w_in_end : input_width;
  const int h_start = h_in_start > 0 ? h_in_start : 0;
  const int w_start = w_in_start > 0 ? w_in_start : 0;
  int weight_offset = 0;

#pragma unroll
  for (int h_in = h_in_start; h_in < h_in_end; h_in += dilate_height) {
#pragma unroll
    for (int w_in = w_in_start; w_in < w_in_end; w_in += dilate_width) {
      if (h_in >= h_start && h_in < h_end && w_in >= w_start && w_in < w_end) {
        int offset = in_offset + h_in * input_width + w_in;
        T in_data = input_data[offset];
        if (fuse_relu_before_conv) {
          value += weight[weight_offset] * max(0.0f, in_data);
        } else {
          value += weight[weight_offset] * in_data;
        }
      }
      weight_offset++;
    }
  }
  int index = batch * output_channels * output_height * output_width +
              c_out * output_height * output_width + h_out * output_width +
              w_out;
  output_data[index] = value;
}

// A Cuda kernel to compute the depthwise convolution forward pass
// in NHWC format.
template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvNHWC(
    ARG_DEFINE_KernelDepthwiseConv) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= (output_channels * batch_size * output_height * output_width))
    return;

  const int c_out = idx % output_channels;
  const int w_out = (idx / output_channels) % output_width;
  const int h_out = (idx / output_channels / output_width) % output_height;
  const int batch = idx / output_width / output_height / output_channels;

  const int c_in = c_out / filter_multiplier;
  const T* weight = filter_data + c_out * filter_height * filter_width;
  T value = 0;
  const int h_in_start = -padding_height + h_out * stride_height;
  const int w_in_start = -padding_width + w_out * stride_width;
  const int h_in_end = h_in_start + filter_height * dilate_height;
  const int w_in_end = w_in_start + filter_width * dilate_width;

  const int h_end = h_in_end < input_height ? h_in_end : input_height;
  const int w_end = w_in_end < input_width ? w_in_end : input_width;
  const int h_start = h_in_start > 0 ? h_in_start : 0;
  const int w_start = w_in_start > 0 ? w_in_start : 0;
  int weight_offset = 0;

#pragma unroll
  for (int h_in = h_in_start; h_in < h_in_end; h_in += dilate_height) {
#pragma unroll
    for (int w_in = w_in_start; w_in < w_in_end; w_in += dilate_width) {
      if (h_in >= h_start && h_in < h_end && w_in >= w_start && w_in < w_end) {
        int offset = ((batch * input_height + h_in) * input_width + w_in) *
                         output_channels +
                     c_in;
        T in_data = input_data[offset];
        if (fuse_relu_before_conv) {
          value += weight[weight_offset] * max(0.0f, in_data);
        } else {
          value += weight[weight_offset] * in_data;
        }
      }
      weight_offset++;
    }
  }
  int index = batch * output_channels * output_height * output_width +
              h_out * output_width * output_channels + w_out * output_channels +
              c_out;
  output_data[index] = value;
}

template <typename T, int c_filter, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvCFilter(
    ARG_DEFINE_KernelDepthwiseConv) {
  const int kWeghtSize = c_filter * c_filter;
  T r_weight[kWeghtSize];
  const int batch = blockIdx.y;
  const int c_out = blockIdx.x;
  const T* weight = filter_data + c_out * c_filter * c_filter;
  for (int i = 0; i < c_filter * c_filter; i++) r_weight[i] = weight[i];

  for (int w_out = threadIdx.x; w_out < output_width; w_out += blockDim.x) {
    for (int h_out = threadIdx.y; h_out < output_height; h_out += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_out = blockIdx.x;

      const int c_in = c_out / filter_multiplier;
      T value = 0;
      const int h_in_start = -padding_height + h_out * stride_height;
      const int w_in_start = -padding_width + w_out * stride_width;
      const int h_in_end = h_in_start + c_filter * dilate_height;
      const int w_in_end = w_in_start + c_filter * dilate_width;

      int in_offset;
      if (data_layout != DataLayout::kNHWC) {
        in_offset =
            ((batch * input_channels + c_in) * input_height) * input_width;
      } else {
        in_offset = batch * input_height * input_width * input_channels;
      }

      const int h_end = h_in_end < input_height ? h_in_end : input_height;
      const int w_end = w_in_end < input_width ? w_in_end : input_width;
      const int h_start = h_in_start > 0 ? h_in_start : 0;
      const int w_start = w_in_start > 0 ? w_in_start : 0;

      for (int h_in = h_in_start, h_f = 0; h_f < c_filter;
           h_in += dilate_height, h_f++) {
        for (int w_in = w_in_start, w_f = 0; w_f < c_filter;
             w_in += dilate_width, w_f++) {
          if (h_in >= 0 && h_in < input_height && w_in >= 0 &&
              w_in < input_width) {
            int offset;
            if (data_layout != DataLayout::kNHWC) {
              offset = in_offset + h_in * input_width + w_in;
            } else {
              offset = in_offset +
                       (h_in * input_width + w_in) * input_channels + c_in;
            }
            if (fuse_relu_before_conv) {
              value += r_weight[h_f * c_filter + w_f] *
                       max(0.0f, input_data[offset]);
            } else {
              value += r_weight[h_f * c_filter + w_f] * input_data[offset];
            }
          }
        }
      }
      int index;
      if (data_layout != DataLayout::kNHWC) {
        index = ((batch * gridDim.x + c_out) * output_height + h_out) *
                    output_width +
                w_out;
      } else {
        index = ((batch * output_height + h_out) * output_width + w_out) *
                    gridDim.x +
                c_out;
      }
      output_data[index] = value;
    }
  }
}

template <typename T, int c_filter_multiplier, int c_stride, int c_filter,
          bool fuse_relu_before_conv>
__global__ void KernelDepthwiseConvSp(ARG_DEFINE_KernelDepthwiseConv) {
  int final_filter_multiplier = filter_multiplier;
  int h_stride = stride_height;
  int w_stride = stride_width;
  if (c_filter_multiplier != 0) {
    final_filter_multiplier = c_filter_multiplier;
    h_stride = c_stride;
    w_stride = c_stride;
  }
  if (c_filter == -1) {
    if (data_layout == DataLayout::kNCHW) {
      KernelDepthwiseConvNCHW<T, fuse_relu_before_conv>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          final_filter_multiplier, filter_height, filter_width, h_stride,
          w_stride, padding_height, padding_width, dilate_height, dilate_width,
          output_data, data_layout);
    } else {
      KernelDepthwiseConvNHWC<T, fuse_relu_before_conv>(
          input_data, filter_data, batch_size, output_channels, output_height,
          output_width, input_channels, input_height, input_width,
          final_filter_multiplier, filter_height, filter_width, h_stride,
          w_stride, padding_height, padding_width, dilate_height, dilate_width,
          output_data, data_layout);
    }
  } else {
    KernelDepthwiseConvCFilter<T, c_filter, fuse_relu_before_conv>(
        input_data, filter_data, batch_size, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        final_filter_multiplier, filter_height, filter_width, h_stride,
        w_stride, padding_height, padding_width, dilate_height, dilate_width,
        output_data, data_layout);
  }
}

// CUDA kernel to compute the depthwise convolution backprop w.r.t input.
#define ARG_DEFINE_KernelDepthwiseConvInputGrad                                \
  const T *const input_data, const T *const output_grad_data,                  \
      const T *const filter_data, const int batch_size,                        \
      const int output_channels, const int output_height,                      \
      const int output_width, const int input_channels,                        \
      const int input_height, const int input_width,                           \
      const int filter_multiplier, const int filter_height,                    \
      const int filter_width, const int stride_height, const int stride_width, \
      const int padding_height, const int padding_width,                       \
      const int dilate_height, const int dilate_width,                         \
      T *const input_grad_data,                                                \
      const DataLayout data_layout = DataLayout::kNCHW

template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvInputGrad(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  for (int w_in = threadIdx.x; w_in < input_width; w_in += blockDim.x) {
    for (int h_in = threadIdx.y; h_in < input_height; h_in += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_in = blockIdx.x;

      const int c_out_start = c_in * filter_multiplier;

      int h_out_start =
          h_in - (filter_height - 1) * dilate_height + padding_height;

      int h_out_end = h_in + padding_height;

      int w_out_start =
          w_in - (filter_width - 1) * dilate_width + padding_width;

      int w_out_end = w_in + padding_width;

      T value = 0;
      int index;
      if (data_layout != DataLayout::kNHWC) {
        index =
            ((batch * gridDim.x + c_in) * input_height + h_in) * input_width +
            w_in;
      } else {
        index =
            ((batch * input_height + h_in) * input_width + w_in) * gridDim.x +
            c_in;
      }

      if (fuse_relu_before_conv) {
        if (input_data[index] <= 0) {
          input_grad_data[index] = 0;
          continue;
        }
      }

      for (int c_out = c_out_start; c_out < c_out_start + filter_multiplier;
           c_out++) {
        int filter_offset = (c_out + 1) * filter_height * filter_width;
        for (int h_out = h_out_start; h_out <= h_out_end;
             h_out += dilate_height) {
          for (int w_out = w_out_start; w_out <= w_out_end;
               w_out += dilate_width) {
            filter_offset--;
            int s_h_out = h_out / stride_height;
            int s_w_out = w_out / stride_width;
            if (h_out % stride_height == 0 && w_out % stride_width == 0 &&
                s_h_out >= 0 && s_h_out < output_height && s_w_out >= 0 &&
                s_w_out < output_width) {
              int output_grad_offset;
              if (data_layout != DataLayout::kNHWC) {
                output_grad_offset =
                    ((batch * output_channels + c_out) * output_height +
                     s_h_out) *
                        output_width +
                    s_w_out;
              } else {
                output_grad_offset =
                    ((batch * output_height + s_h_out) * output_width +
                     s_w_out) *
                        output_channels +
                    c_out;
              }
              value += output_grad_data[output_grad_offset] *
                       filter_data[filter_offset];
            }
          }
        }
      }
      input_grad_data[index] = value;
    }
  }
}

template <typename T, int c_filter, int c_filter_multiplier,
          bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvInputGradCFilter(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  const int kWeghtSize = c_filter * c_filter * c_filter_multiplier + 1;
  T r_weight[kWeghtSize];
  const int batch = blockIdx.y;
  const int c_in = blockIdx.x;

  for (int c_i = 0; c_i < filter_multiplier; c_i++) {
    int c_out = c_in * filter_multiplier + c_i;
    const T* weight = filter_data + c_out * c_filter * c_filter;
    for (int i = 0; i < c_filter * c_filter; i++)
      r_weight[i + c_i * c_filter * c_filter] =
          weight[c_filter * c_filter - i - 1];
  }

  for (int w_in = threadIdx.x; w_in < input_width; w_in += blockDim.x) {
    for (int h_in = threadIdx.y; h_in < input_height; h_in += blockDim.y) {
      const int batch = blockIdx.y;
      const int c_in = blockIdx.x;

      int h_out_start = h_in - (c_filter - 1) * dilate_height + padding_height;

      int w_out_start = w_in - (c_filter - 1) * dilate_width + padding_width;

      T value = 0;
      int index;
      if (data_layout != DataLayout::kNHWC) {
        index =
            ((batch * gridDim.x + c_in) * input_height + h_in) * input_width +
            w_in;
      } else {
        index =
            ((batch * input_height + h_in) * input_width + w_in) * gridDim.x +
            c_in;
      }
      if (fuse_relu_before_conv) {
        if (input_data[index] <= 0) {
          input_grad_data[index] = 0;
          continue;
        }
      }

      for (int c_i = 0; c_i < filter_multiplier; c_i++) {
        int c_out = c_in * filter_multiplier + c_i;
        for (int h_out = h_out_start, h_f = 0; h_f < c_filter;
             h_out += dilate_height, h_f++) {
          for (int w_out = w_out_start, w_f = 0; w_f < c_filter;
               w_out += dilate_width, w_f++) {
            int s_h_out = h_out / stride_height;
            int s_w_out = w_out / stride_width;
            if (h_out % stride_height == 0 && w_out % stride_width == 0 &&
                s_h_out >= 0 && s_h_out < output_height && s_w_out >= 0 &&
                s_w_out < output_width) {
              int output_grad_offset;
              if (data_layout != DataLayout::kNHWC) {
                output_grad_offset =
                    ((batch * output_channels + c_out) * output_height +
                     s_h_out) *
                        output_width +
                    s_w_out;
              } else {
                output_grad_offset =
                    ((batch * output_height + s_h_out) * output_width +
                     s_w_out) *
                        output_channels +
                    c_out;
              }
              value +=
                  output_grad_data[output_grad_offset] *
                  r_weight[h_f * c_filter + w_f + c_i * c_filter * c_filter];
            }
          }
        }
      }
      input_grad_data[index] = value;
    }
  }
}

template <typename T, int c_filter_multiplier, int c_stride, int c_filter,
          bool fuse_relu_before_conv>
__global__ void KernelDepthwiseConvInputGradSp(
    ARG_DEFINE_KernelDepthwiseConvInputGrad) {
  if (c_filter_multiplier == 0)
    KernelDepthwiseConvInputGrad<T, fuse_relu_before_conv>(
        input_data, output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, input_grad_data, data_layout);
  else if (c_filter == -1)
    KernelDepthwiseConvInputGrad<T, fuse_relu_before_conv>(
        input_data, output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, c_stride, c_stride,
        padding_height, padding_width, dilate_height, dilate_width,
        input_grad_data, data_layout);
  else
    KernelDepthwiseConvInputGradCFilter<T, c_filter, c_filter_multiplier,
                                        fuse_relu_before_conv>(
        input_data, output_grad_data, filter_data, batch_size, output_channels,
        output_height, output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, c_stride, c_stride,
        padding_height, padding_width, dilate_height, dilate_width,
        input_grad_data, data_layout);
}

// Cuda kernel to compute the depthwise convolution backprop w.r.t. filter.
template <typename T, bool fuse_relu_before_conv>
__device__ __inline__ void KernelDepthwiseConvFilterGrad(
    const T* output_grad_data, const T* input_data, const int num,
    const int output_channels, const int output_height, const int output_width,
    const int input_channels, const int input_height, const int input_width,
    const int filter_multiplier, const int filter_height,
    const int filter_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, const int dilate_height,
    const int dilate_width, T* filter_grad_data,
    const DataLayout data_layout = DataLayout::kNCHW) {
  T s = 0;

  int gbid = ((blockIdx.z * gridDim.y) + blockIdx.y) * gridDim.x + blockIdx.x;

  for (int image_w = threadIdx.x; image_w < output_width;
       image_w += blockDim.x) {
    for (int bid = 0; bid < num; bid++) {
      for (int image_h = threadIdx.y; image_h < output_height;
           image_h += blockDim.y) {
        int kernel_id = blockIdx.z;
        int kernel_h = blockIdx.y * dilate_height - padding_height;
        int kernel_w = blockIdx.x * dilate_width - padding_width;

        int image_hk = image_h * stride_height + kernel_h;
        int image_wk = image_w * stride_width + kernel_w;
        if (image_hk < 0 || image_hk >= input_height) continue;
        if (image_wk < 0 || image_wk >= input_width) continue;
#define gaid(N, C, H, W) \
  ((((N)*gridDim.z + (C)) * output_height + (H)) * output_width + (W))
#define gaid_nhwc(N, H, W, C) \
  ((((N)*output_height + (H)) * output_width + (W)) * gridDim.z + (C))
        int input_id;
        if (data_layout != DataLayout::kNHWC) {
          input_id = ((bid * (gridDim.z / filter_multiplier) +
                       kernel_id / filter_multiplier) *
                          input_height +
                      image_hk) *
                         input_width +
                     image_wk;
          if (fuse_relu_before_conv) {
            s += output_grad_data[gaid(bid, kernel_id, image_h, image_w)] *
                 max(0.0f, input_data[input_id]);
          } else {
            s += output_grad_data[gaid(bid, kernel_id, image_h, image_w)] *
                 input_data[input_id];
          }
        } else {
          input_id =
              ((bid * input_height + image_hk) * input_width + image_wk) *
                  (gridDim.z / filter_multiplier) +
              kernel_id / filter_multiplier;
          if (fuse_relu_before_conv) {
            s += output_grad_data[gaid_nhwc(bid, image_h, image_w, kernel_id)] *
                 max(0.0f, input_data[input_id]);
          } else {
            s += output_grad_data[gaid_nhwc(bid, image_h, image_w, kernel_id)] *
                 input_data[input_id];
          }
        }

#undef gaid
      }
    }
  }
  CudaAtomicAddWithWarp(&filter_grad_data[gbid], s);
}

template <typename T, int c_filter_multiplier, bool fuse_relu_before_conv>
__global__ void KernelDepthwiseConvFilterGradSp(
    const T* output_grad_data, const T* input_data, const int num,
    const int output_channels, const int output_height, const int output_width,
    const int input_channels, const int input_height, const int input_width,
    const int filter_multiplier, const int filter_height,
    const int filter_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, const int dilate_height,
    const int dilate_width, T* filter_grad_data,
    const DataLayout data_layout = DataLayout::kNCHW) {
  if (c_filter_multiplier == 0)
    KernelDepthwiseConvFilterGrad<T, fuse_relu_before_conv>(
        output_grad_data, input_data, num, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, filter_grad_data, data_layout);
  else
    KernelDepthwiseConvFilterGrad<T, fuse_relu_before_conv>(
        output_grad_data, input_data, num, output_channels, output_height,
        output_width, input_channels, input_height, input_width,
        c_filter_multiplier, filter_height, filter_width, stride_height,
        stride_width, padding_height, padding_width, dilate_height,
        dilate_width, filter_grad_data, data_layout);
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <class T, bool fuse_relu_before_conv>
class DepthwiseConvFunctor<platform::CUDADeviceContext, T,
                           fuse_relu_before_conv> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& filter,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations, framework::Tensor* output,
                  const DataLayout data_layout = DataLayout::kNCHW) {
    const int batch_size = input.dims()[0];
    const int input_channels =
        (data_layout != DataLayout::kNHWC ? input.dims()[1] : input.dims()[3]);
    const int input_height =
        (data_layout != DataLayout::kNHWC ? input.dims()[2] : input.dims()[1]);
    const int input_width =
        (data_layout != DataLayout::kNHWC ? input.dims()[3] : input.dims()[2]);
    const int output_channels =
        (data_layout != DataLayout::kNHWC ? output->dims()[1]
                                          : output->dims()[3]);
    const int output_height =
        (data_layout != DataLayout::kNHWC ? output->dims()[2]
                                          : output->dims()[1]);
    const int output_width =
        (data_layout != DataLayout::kNHWC ? output->dims()[3]
                                          : output->dims()[2]);
    const int ksize_height = filter.dims()[2];
    const int ksize_width = filter.dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* filter_data = filter.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int thread = 512;
    if (output_width > 1024 && output_width <= 2048)
      thread = (output_width - 1) / 2 + 1;
    else if (output_width > 512 && output_width <= 1024)
      thread = output_width;
    int blocks = std::min(std::max(thread / output_width, 1), output_height);
    dim3 threads(std::min(output_width, thread), blocks, 1);
    dim3 grid(output_channels, batch_size, 1);
    int filter_multiplier = output_channels / input_channels;

    int nums_output =
        batch_size * output_channels * output_height * output_width;
    int block_size = 512;

#define check_case(c_filter_multiplier, c_stride, c_filter)                  \
  if (c_filter_multiplier == 0 ||                                            \
      filter_multiplier == c_filter_multiplier &&                            \
          stride_height == stride_width && stride_height == c_stride &&      \
          (ksize_height == ksize_width && ksize_height == c_filter ||        \
           c_filter == -1)) {                                                \
    if (c_filter == -1) {                                                    \
      threads.x = block_size;                                                \
      grid.x = (nums_output + block_size - 1) / block_size;                  \
      threads.y = threads.z = grid.y = grid.z = 1;                           \
    }                                                                        \
    KernelDepthwiseConvSp<                                                   \
        T, c_filter_multiplier, c_stride, c_filter,                          \
        fuse_relu_before_conv><<<grid, threads, 0, context.stream()>>>(      \
        input_data, filter_data, batch_size, output_channels, output_height, \
        output_width, input_channels, input_height, input_width,             \
        filter_multiplier, ksize_height, ksize_width, stride_height,         \
        stride_width, padding_height, padding_width, dilate_height,          \
        dilate_width, output_data, data_layout);                             \
    return;                                                                  \
  }
    check_case(1, 1, 3);
    check_case(1, 1, 5);
    check_case(1, 1, -1);
    check_case(1, 2, 3);
    check_case(1, 2, 5);
    check_case(1, 2, -1);
    check_case(2, 1, 3);
    check_case(2, 1, 5);
    check_case(2, 1, -1);
    check_case(2, 2, 3);
    check_case(2, 2, 5);
    check_case(2, 2, -1);
    check_case(0, 0, -1);
// NOTE(liangdun): 0,0 for other case
// add other case if needed, e.g. check_case(2^n,1)
#undef check_case
  }
};

template <typename T, bool fuse_relu_before_conv>
class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, T,
                                    fuse_relu_before_conv> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& filter,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* input_grad,
                  const DataLayout data_layout = DataLayout::kNCHW) {
    const int batch_size = input.dims()[0];
    const int input_channels =
        (data_layout != DataLayout::kNHWC ? input.dims()[1] : input.dims()[3]);
    const int input_height =
        (data_layout != DataLayout::kNHWC ? input.dims()[2] : input.dims()[1]);
    const int input_width =
        (data_layout != DataLayout::kNHWC ? input.dims()[3] : input.dims()[2]);
    const int output_channels =
        (data_layout != DataLayout::kNHWC ? output_grad.dims()[1]
                                          : output_grad.dims()[3]);
    const int output_height =
        (data_layout != DataLayout::kNHWC ? output_grad.dims()[2]
                                          : output_grad.dims()[1]);
    const int output_width =
        (data_layout != DataLayout::kNHWC ? output_grad.dims()[3]
                                          : output_grad.dims()[2]);
    const int ksize_height = filter.dims()[2];
    const int ksize_width = filter.dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* filter_data = filter.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int thread = 512;
    if (input_width > 1024 && input_width <= 2048)
      thread = (input_width - 1) / 2 + 1;
    else if (input_width > 512 && input_width <= 1024)
      thread = input_width;
    int blocks = std::min(std::max(thread / input_width, 1), input_height);
    dim3 threads(std::min(input_width, thread), blocks, 1);
    dim3 grid(input_channels, batch_size, 1);
    int filter_multiplier = output_channels / input_channels;

#define check_case(c_filter_multiplier, c_stride, c_filter)             \
  if (c_filter_multiplier == 0 ||                                       \
      filter_multiplier == c_filter_multiplier &&                       \
          stride_height == stride_width && stride_height == c_stride && \
          (ksize_height == ksize_width && ksize_height == c_filter ||   \
           c_filter == -1)) {                                           \
    KernelDepthwiseConvInputGradSp<                                     \
        T, c_filter_multiplier, c_stride, c_filter,                     \
        fuse_relu_before_conv><<<grid, threads, 0, context.stream()>>>( \
        input_data, output_grad_data, filter_data, batch_size,          \
        output_channels, output_height, output_width, input_channels,   \
        input_height, input_width, filter_multiplier, ksize_height,     \
        ksize_width, stride_height, stride_width, padding_height,       \
        padding_width, dilate_height, dilate_width, input_grad_data,    \
        data_layout);                                                   \
    return;                                                             \
  }
    check_case(1, 1, 3);
    check_case(1, 1, 5);
    check_case(1, 1, -1);
    check_case(1, 2, 3);
    check_case(1, 2, 5);
    check_case(1, 2, -1);
    check_case(2, 1, 3);
    check_case(2, 1, 5);
    check_case(2, 1, -1);
    check_case(2, 2, 3);
    check_case(2, 2, 5);
    check_case(2, 2, -1);
    check_case(0, 0, -1);
// NOTE(liangdun): 0,0 for other case
// add other case if needed, e.g. check_case(2^n,1)
#undef check_case
  }
};

template <typename T, bool fuse_relu_before_conv>
class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext, T,
                                     fuse_relu_before_conv> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  framework::Tensor* filter_grad,
                  const DataLayout data_layout = DataLayout::kNCHW) {
    const int batch_size = input.dims()[0];
    const int input_channels =
        (data_layout != DataLayout::kNHWC ? input.dims()[1] : input.dims()[3]);
    const int input_height =
        (data_layout != DataLayout::kNHWC ? input.dims()[2] : input.dims()[1]);
    const int input_width =
        (data_layout != DataLayout::kNHWC ? input.dims()[3] : input.dims()[2]);
    const int output_channels =
        (data_layout != DataLayout::kNHWC ? output_grad.dims()[1]
                                          : output_grad.dims()[3]);
    const int output_height =
        (data_layout != DataLayout::kNHWC ? output_grad.dims()[2]
                                          : output_grad.dims()[1]);
    const int output_width =
        (data_layout != DataLayout::kNHWC ? output_grad.dims()[3]
                                          : output_grad.dims()[2]);
    const int ksize_height = filter_grad->dims()[2];
    const int ksize_width = filter_grad->dims()[3];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];
    const int dilate_height = dilations[0];
    const int dilate_width = dilations[1];

    const T* input_data = input.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* filter_grad_data = filter_grad->mutable_data<T>(context.GetPlace());

    int block_size = 512;
    if (output_width > 1024 && output_width <= 2048)
      block_size = (output_width - 1) / 2 + 1;
    else if (output_width > 512 && output_width <= 1024)
      block_size = output_width;
    int crop_output_height =
        std::min(std::max(block_size / output_width, 1), output_height);
    dim3 grid(ksize_width, ksize_height, output_channels);
    dim3 threads(std::min(output_width, block_size), crop_output_height, 1);
    int filter_multiplier = output_channels / input_channels;

#define check_case(c_filter_multiplier)                                       \
  if (c_filter_multiplier == 0 || c_filter_multiplier == filter_multiplier) { \
    KernelDepthwiseConvFilterGradSp<                                          \
        T, c_filter_multiplier,                                               \
        fuse_relu_before_conv><<<grid, threads, 0, context.stream()>>>(       \
        output_grad_data, input_data, batch_size, output_channels,            \
        output_height, output_width, input_channels, input_height,            \
        input_width, filter_multiplier, ksize_height, ksize_width,            \
        stride_height, stride_width, padding_height, padding_width,           \
        dilate_height, dilate_width, filter_grad_data, data_layout);          \
    return;                                                                   \
  }
    check_case(1);
    check_case(0);
#undef check_case
  }
};

template class DepthwiseConvFunctor<platform::CUDADeviceContext, float, false>;
template class DepthwiseConvFunctor<platform::CUDADeviceContext, double, false>;

template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, float,
                                             false>;
template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext,
                                             double, false>;

template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              float, false>;
template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              double, false>;

template class DepthwiseConvFunctor<platform::CUDADeviceContext, float, true>;
template class DepthwiseConvFunctor<platform::CUDADeviceContext, double, true>;

template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext, float,
                                             true>;
template class DepthwiseConvInputGradFunctor<platform::CUDADeviceContext,
                                             double, true>;

template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              float, true>;
template class DepthwiseConvFilterGradFunctor<platform::CUDADeviceContext,
                                              double, true>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
