#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/prelu.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {
namespace math {

#define CUDA_NUM_THREADS 1024

inline static int PADDLE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__global__ void PReluChannelFirstWiseKernel(const T *input, const T *alpha,
                                            T *output, size_t channel_num,
                                            size_t plane_size, size_t numel) {
  CUDA_KERNEL_LOOP(index, numel) {
    size_t temp = index / plane_size;
    size_t channel_index = temp % channel_num;
    T scale = alpha[channel_index];
    T x = input[index];
    T zero = static_cast<T>(0);
    output[index] = (x > zero) ? x : scale * x;
  }
}

template <typename T>
__global__ void PReluChannelLastWiseKernel(const T *input, const T *alpha,
                                           T *output, size_t channel_num,
                                           size_t numel) {
  CUDA_KERNEL_LOOP(index, numel) {
    size_t channel_index = index % channel_num;
    T scale = alpha[channel_index];
    T x = input[index];
    T zero = static_cast<T>(0);
    output[index] = (x > zero) ? x : scale * x;
  }
}

template <typename T>
__global__ void PReluElementWiseKernel(const T *input, const T *alpha,
                                       T *output, size_t spatial_size,
                                       size_t numel) {
  CUDA_KERNEL_LOOP(index, numel) {
    size_t element_index = index % spatial_size;
    T scale = alpha[element_index];
    T x = input[index];
    T zero = static_cast<T>(0);
    output[index] = (x > zero) ? x : scale * x;
  }
}

template <typename T>
__global__ void PReluScalarKernel(const T *input, const T *alpha, T *output,
                                  size_t numel) {
  T scale = alpha[0];
  CUDA_KERNEL_LOOP(index, numel) {
    T x = input[index];
    T zero = static_cast<T>(0);
    output[index] = (x > zero) ? x : scale * x;
  }
}

template <typename T>
void PreluChannelWiseDirectCUDAFunctor<T>::operator()(
    gpuStream_t stream, const T *input, const T *alpha, T *output,
    size_t batch_size, size_t channel, bool channel_last, size_t numel) {
  if (channel_last) {
    PReluChannelLastWiseKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0,
                                 stream>>>(input, alpha, output, channel,
                                           numel);
  } else {
    PReluChannelFirstWiseKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0,
                                  stream>>>(
        input, alpha, output, channel, numel / batch_size / channel, numel);
  }
}

template <typename T>
void PreluElementWiseDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                                      const T *input,
                                                      const T *alpha, T *output,
                                                      size_t batch_size,
                                                      size_t numel) {
  PReluElementWiseKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0,
                           stream>>>(input, alpha, output, numel / batch_size,
                                     numel);
}

template <typename T>
void PreluScalarDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                                 const T *input, const T *alpha,
                                                 T *output, size_t numel) {
  PReluScalarKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, numel);
}

template class PreluChannelWiseDirectCUDAFunctor<float>;
template class PreluChannelWiseDirectCUDAFunctor<platform::float16>;
template class PreluChannelWiseDirectCUDAFunctor<double>;

template class PreluElementWiseDirectCUDAFunctor<float>;
template class PreluElementWiseDirectCUDAFunctor<platform::float16>;
template class PreluElementWiseDirectCUDAFunctor<double>;

template class PreluScalarDirectCUDAFunctor<float>;
template class PreluScalarDirectCUDAFunctor<platform::float16>;
template class PreluScalarDirectCUDAFunctor<double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
