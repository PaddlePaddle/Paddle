#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/prelu.h"

namespace paddle {
namespace operators {
namespace math {

static const int CUDA_NUM_THREADS = 1024;
static const int CUDA_MAX_NUM_BLOCKS = 65535;
inline static int GET_NUM_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__global__ void PReluChannelWiseKernel(const T *input, const T *alpha,
                                       T *output, int channel,
                                       size_t spatial_size) {
  size_t offset = blockIdx.x * spatial_size;
  const T *in = input + offset;
  T *out = output + offset;
  T scale = alpha[blockIdx.x % channel];

  for (size_t i = threadIdx.x; i < spatial_size; i += blockDim.x) {
    T x = in[i];
    out[i] = (x > 0) ? x : scale * x;
  }
}

template <typename T>
__global__ void PReluElementWiseKernel(const T *input, const T *alpha,
                                       T *output, size_t spatial_size) {
  size_t offset = blockIdx.x * spatial_size;
  const T *in = input + offset;
  const T *scale = alpha + offset;
  T *out = output + offset;

  for (size_t i = threadIdx.x; i < spatial_size; i += blockDim.x) {
    T x = in[i];
    out[i] = (x > 0) ? x : scale[i] * x;
  }
}

template <typename T>
__global__ void PReluScalarKernel(const T *input, const T *alpha, T *output,
                                  size_t spatial_size) {
  size_t offset = blockIdx.x * spatial_size;
  const T *in = input + offset;
  T scale = *alpha;
  T *out = output + offset;

  for (size_t i = threadIdx.x; i < spatial_size; i += blockDim.x) {
    T x = in[i];
    out[i] = (x > 0) ? x : scale * x;
  }
}

template <typename T>
static inline void PReluChannelWise(hipStream_t stream, const T *input,
                                    const T *alpha, T *output,
                                    std::vector<int> input_shape) {
  size_t unroll = input_shape[0] * input_shape[1];
  size_t spatial_size = input_shape[2] * input_shape[3];
  CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
  PReluChannelWiseKernel<<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, input_shape[1], spatial_size);
}

template <typename T>
static inline void PReluElementWise(hipStream_t stream, const T *input,
                                    const T *alpha, T *output,
                                    std::vector<int> input_shape) {
  size_t unroll = input_shape[0] * input_shape[1];
  size_t spatial_size = input_shape[2] * input_shape[3];
  CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
  PReluElementWiseKernel<<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, spatial_size);
}

template <typename T>
static inline void PReluScalar(hipStream_t stream, const T *input,
                               const T *alpha, T *output,
                               std::vector<int> input_shape) {
  size_t unroll = input_shape[0] * input_shape[1];
  size_t spatial_size = input_shape[2] * input_shape[3];
  CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
  PReluScalarKernel<<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, spatial_size);
}

template <typename T>
void PreluChannelWiseDirectCUDAFunctor<T>::operator()(
    hipStream_t stream, const T *input, const T *alpha, T *output,
    std::vector<int> input_shape) {
  size_t unroll = input_shape[0] * input_shape[1];
  size_t spatial_size = input_shape[2] * input_shape[3];
  CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
  PReluChannelWiseKernel<<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, input_shape[1], spatial_size);
}

template <typename T>
void PreluElementWiseDirectCUDAFunctor<T>::operator()(
    hipStream_t stream, const T *input, const T *alpha, T *output,
    std::vector<int> input_shape) {
  size_t unroll = input_shape[0] * input_shape[1];
  size_t spatial_size = input_shape[2] * input_shape[3];
  CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
  PReluElementWiseKernel<<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, spatial_size);
}

template <typename T>
void PreluScalarDirectCUDAFunctor<T>::operator()(hipStream_t stream,
                                                 const T *input, const T *alpha,
                                                 T *output,
                                                 std::vector<int> input_shape) {
  size_t unroll = input_shape[0] * input_shape[1];
  size_t spatial_size = input_shape[2] * input_shape[3];
  CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
  PReluScalarKernel<<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, spatial_size);
}

template class PreluChannelWiseDirectCUDAFunctor<float>;
template class PreluChannelWiseDirectCUDAFunctor<double>;

template class PreluElementWiseDirectCUDAFunctor<float>;
template class PreluElementWiseDirectCUDAFunctor<double>;

template class PreluScalarDirectCUDAFunctor<float>;
template class PreluScalarDirectCUDAFunctor<double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
