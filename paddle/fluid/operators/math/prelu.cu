#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/prelu.h"

namespace paddle {
namespace operators {
namespace math {

#define CUDA_NUM_THREADS 1024

inline static int PADDLE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__global__ void PReluChannelWiseKernel(const T *input, const T *alpha,
                                       T *output, size_t channel_num,
                                       size_t plane_size, size_t numel) {
  CUDA_KERNEL_LOOP(index, numel) {
    size_t temp = index / plane_size;
    size_t channel_index = temp % channel_num;
    T scale = alpha[channel_index];
    T x = input[index];
    output[index] = (x > 0) ? x : scale * x;
  }
}

template <typename T>
__global__ void PReluElementWiseKernel(const T *input, const T *alpha,
                                       T *output, size_t spatial_size,
                                       size_t numel) {
  CUDA_KERNEL_LOOP(index, numel) {
    size_t element_index = index % spatial_size;
    T scale = alpha[element_index];
    T x = input[index];
    output[index] = (x > 0) ? x : scale * x;
  }
}

template <typename T>
__global__ void PReluScalarKernel(const T *input, const T *alpha, T *output,
                                  size_t numel) {
  T scale = alpha[0];
  CUDA_KERNEL_LOOP(index, numel) {
    T x = input[index];
    output[index] = (x > 0) ? x : scale * x;
  }
}

template <typename T>
void PreluChannelWiseDirectCUDAFunctor<T>::operator()(
    gpuStream_t stream, const T *input, const T *alpha, T *output,
    size_t batch_size, size_t channel, size_t numel) {
  PReluChannelWiseKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0,
                           stream>>>(input, alpha, output, channel,
                                     numel / batch_size / channel, numel);
}

template <typename T>
void PreluElementWiseDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                                      const T *input,
                                                      const T *alpha, T *output,
                                                      size_t batch_size,
                                                      size_t numel) {
  PReluElementWiseKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0,
                           stream>>>(input, alpha, output, numel / batch_size,
                                     numel);
}

template <typename T>
void PreluScalarDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                                 const T *input, const T *alpha,
                                                 T *output, size_t numel) {
  PReluScalarKernel<<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
      input, alpha, output, numel);
}

template class PreluChannelWiseDirectCUDAFunctor<float>;
template class PreluChannelWiseDirectCUDAFunctor<double>;

template class PreluElementWiseDirectCUDAFunctor<float>;
template class PreluElementWiseDirectCUDAFunctor<double>;

template class PreluScalarDirectCUDAFunctor<float>;
template class PreluScalarDirectCUDAFunctor<double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
