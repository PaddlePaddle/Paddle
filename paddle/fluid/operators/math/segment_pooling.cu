#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/segment_pooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T, typename Index, int DimTileSize>
__global__ void SegmentMeanCustomKernel(
    const Index* segment_ids, const T* input, T* output, T* summed_ids,
    const Index input_length_size, const Index inner_dim_size,
    const Index output_length_size, const Index total_stripe_count) {
  CUDA_KERNEL_LOOP(stripe_index, total_stripe_count) {
    const Index segment_offset = stripe_index % inner_dim_size;
    const Index dim_index_base =
        stripe_index / inner_dim_size * Index(DimTileSize);
    const Index actual_height =
        min(Index(DimTileSize), input_length_size - dim_index_base);

    Index first_segment_id = segment_ids[dim_index_base];
    Index last_segment_id = -1;
    if (dim_index_base > 0) {
      last_segment_id = segment_ids[dim_index_base - 1];
    }
    if (segment_offset == 0) {
      T sum = T(0);
      for (Index j = 0; j < actual_height; j++) {
        Index current_segment_id = segment_ids[dim_index_base + j];
        // Note(ZHUI): following check may cause
        // hipErrorLaunchOutOfResources.
        // PADDLE_ENFORCE(current_segment_id >= last_segment_id,
        //               "the segment ids should be sorted, but got "
        //               "segment_ids[%d]:%d > segment_ids[%d]:%d.",
        //               dim_index_base + j - 1, dim_index_base + j,
        //               last_segment_id, current_segment_id);

        if (j > 0 && current_segment_id > last_segment_id) {
          if (last_segment_id == first_segment_id) {
            platform::CudaAtomicAdd(summed_ids + last_segment_id, sum);
          } else {
            *(summed_ids + last_segment_id) = sum;
          }
          sum = T(0);
        }
        sum += T(1);
        last_segment_id = current_segment_id;
      }
      platform::CudaAtomicAdd(summed_ids + last_segment_id, sum);
    }
    // ensure last_segment_id is the largest
    last_segment_id = output_length_size;
    __syncthreads();
    T sum = T(0);
    for (Index j = 0; j < actual_height; j++) {
      Index current_segment_id = segment_ids[dim_index_base + j];
      if (current_segment_id > last_segment_id) {
        const Index output_index =
            last_segment_id * inner_dim_size + segment_offset;
        if (last_segment_id == first_segment_id) {
          platform::CudaAtomicAdd(output + output_index,
                                  sum / *(summed_ids + last_segment_id));
        } else {
          *(output + output_index) = sum / *(summed_ids + last_segment_id);
        }
        sum = T(0);
      }
      sum += input[(dim_index_base + j) * inner_dim_size + segment_offset];
      last_segment_id = current_segment_id;
    }
    const Index output_index =
        last_segment_id * inner_dim_size + segment_offset;
    platform::CudaAtomicAdd(output + output_index,
                            sum / *(summed_ids + last_segment_id));
  }
}

template <typename T, typename Index, typename Helper, typename Pool>
__global__ void SegmentOpsKernel(const Index* segment_ids, const T* input,
                                 T* output, Helper h, Pool pool) {
  CUDA_KERNEL_LOOP(stripe_index, h.total_stripe_count) {
    Index segment_offset, dim_index_base, actual_height;
    Index inner_dim_size = h.inner_dim_size;
    h.calculate(stripe_index, &segment_offset, &dim_index_base, &actual_height);

    T minmax = pool.initial();
    Index first_segment_id = segment_ids[dim_index_base];
    // -1 is for the start value when interval_id = 0
    Index last_segment_id = -1;
    if (dim_index_base > 0) {
      last_segment_id = segment_ids[dim_index_base - 1];
    }

    for (Index j = 0; j < actual_height; j++) {
      Index current_segment_id = segment_ids[dim_index_base + j];
      // ensure the segment_ids is sorted.
      PADDLE_ENFORCE(current_segment_id >= last_segment_id,
                     "The segment ids should be sorted, but got "
                     "segment_ids[%d]:%d > segment_ids[%d]:%d.",
                     dim_index_base + j - 1, dim_index_base + j,
                     last_segment_id, current_segment_id);

      if (current_segment_id > last_segment_id) {
        // reset the interval value which do not have corresponding ids.
        for (Index interval_id = last_segment_id + 1;
             interval_id < current_segment_id; ++interval_id) {
          *(output + interval_id * inner_dim_size + segment_offset) = 0;
        }
        // don't update result when j=0
        if (j > 0) {
          const Index output_index =
              last_segment_id * inner_dim_size + segment_offset;
          if (last_segment_id == first_segment_id) {
            pool.atomic(output + output_index, minmax);
          } else {
            *(output + output_index) = minmax;
          }
          minmax = pool.initial();
        }
      }
      pool.compute(
          input[(dim_index_base + j) * inner_dim_size + segment_offset],
          &minmax);
      last_segment_id = current_segment_id;
    }
    const Index output_index =
        last_segment_id * inner_dim_size + segment_offset;
    pool.atomic(output + output_index, minmax);
  }
}

template <typename T, typename Index, typename Helper>
__global__ void SegmentIndexGradKernel(const Index* segment_ids, const T* input,
                                       const T* output, const T* out_grad,
                                       T* in_grad, Helper h) {
  CUDA_KERNEL_LOOP(stripe_index, h.total_stripe_count) {
    Index segment_offset, dim_index_base, actual_height;
    h.calculate(stripe_index, &segment_offset, &dim_index_base, &actual_height);

    for (Index j = 0; j < actual_height; j++) {
      Index current_segment_id = segment_ids[dim_index_base + j];
      Index input_index =
          (dim_index_base + j) * h.inner_dim_size + segment_offset;
      Index output_index =
          current_segment_id * h.inner_dim_size + segment_offset;
      if (input[input_index] == output[output_index]) {
        in_grad[input_index] = out_grad[output_index];
      }
    }
  }
}

template <class T>
class MaxPool {
 public:
  DEVICE inline T initial() { return static_cast<T>(-FLT_MAX); }
  DEVICE inline void compute(const T& x, T* y) { *y = *y > x ? *y : x; }
  DEVICE inline T atomic(T* address, const T val) {
    return platform::CudaAtomicMax(address, val);
  }
};

template <class T>
class MinPool {
 public:
  DEVICE inline T initial() { return static_cast<T>(FLT_MAX); }
  DEVICE inline void compute(const T& x, T* y) { *y = *y < x ? *y : x; }
  DEVICE inline T atomic(T* address, const T val) {
    return platform::CudaAtomicMin(address, val);
  }
};

template <class T>
class SumPool {
 public:
  DEVICE inline T initial() { return static_cast<T>(0); }
  DEVICE inline void compute(const T& x, T* y) { *y = *y + x; }
  DEVICE inline T atomic(T* address, const T val) {
    return platform::CudaAtomicAdd(address, val);
  }
};

template <class T>
class ArrangeHelper {
 public:
  const T input_total_size;
  const T input_length_size;
  const T output_length_size;
  T inner_dim_size;
  T total_stripe_count;
  const T DimTileSize = 8;

  ArrangeHelper(T a, T b, T c)
      : input_total_size(a), input_length_size(b), output_length_size(c) {
    T input_outer_dim_num_stripe =
        (input_length_size + DimTileSize - 1) / DimTileSize;
    inner_dim_size = input_total_size / input_length_size;
    total_stripe_count = inner_dim_size * input_outer_dim_num_stripe;
  }

  DEVICE inline void calculate(T stripe_index, T* segment_offset,
                               T* dim_index_base, T* actual_height) {
    *segment_offset = stripe_index % inner_dim_size;
    *dim_index_base = stripe_index / inner_dim_size * DimTileSize;
    *actual_height = min(DimTileSize, input_length_size - *dim_index_base);
  }
};

template <typename T, typename Index>
void SegmentPoolCUDAGradFunctor(const platform::CUDADeviceContext& ctx,
                                const framework::Tensor& input,
                                const framework::Tensor& segment_ids,
                                const framework::Tensor& output,
                                const framework::Tensor& out_grad,
                                framework::Tensor* in_grad,
                                const std::string pooltype = "SUM") {
  auto h = ArrangeHelper<Index>(input.numel(), segment_ids.dims()[0],
                                output.dims()[0]);
  auto config = platform::GetGpuLaunchConfig1D(ctx, h.total_stripe_count);
  if (pooltype == "MAX" || pooltype == "MIN") {
    SegmentIndexGradKernel<T, Index, ArrangeHelper<Index>><<<
        config.block_per_grid.x, config.thread_per_block.x, 0, ctx.stream()>>>(
        segment_ids.data<Index>(), input.data<T>(), output.data<T>(),
        out_grad.data<T>(), in_grad->data<T>(), h);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "Unsupported segment pooling grad operation, Only MAX, MIN "
        "available, but got %s.",
        pooltype));
  }
}

template <typename T>
__global__ void SimpleDiv(T* x, const T* y, const int len, const int dim) {
  for (int i = blockIdx.x; i < len; i += gridDim.x) {
    __shared__ T y_i;
    auto base = i * dim;
    if (threadIdx.x == 0) {
      y_i = y[i];
    }
    __syncthreads();
    for (int j = threadIdx.x; j < dim; j += blockDim.x) {
      x[base + j] /= y_i;
    }
  }
}

template <typename T, typename IndexT>
class SegmentPoolFunctor<platform::CUDADeviceContext, T, IndexT> {
 public:
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& input,
                  const framework::Tensor& segment_ids,
                  framework::Tensor* output,
                  framework::Tensor* summed_ids = nullptr,
                  const std::string pooltype = "SUM") {
    auto h = ArrangeHelper<IndexT>(input.numel(), segment_ids.dims()[0],
                                   output->dims()[0]);
    auto config = platform::GetGpuLaunchConfig1D(ctx, h.total_stripe_count);
    if (pooltype == "MEAN") {
      SegmentMeanCustomKernel<
          T, IndexT, IndexT(8)><<<config.block_per_grid.x,
                                  config.thread_per_block.x, 0, ctx.stream()>>>(
          segment_ids.data<IndexT>(), input.data<T>(), output->data<T>(),
          summed_ids->data<T>(), h.input_length_size, h.inner_dim_size,
          h.output_length_size, h.total_stripe_count);
    } else if (pooltype == "SUM") {
      SumPool<T> pool;
      SegmentOpsKernel<
          T, IndexT, ArrangeHelper<IndexT>,
          SumPool<T>><<<config.block_per_grid.x, config.thread_per_block.x, 0,
                        ctx.stream()>>>(segment_ids.data<IndexT>(),
                                        input.data<T>(), output->data<T>(), h,
                                        pool);
    } else if (pooltype == "MAX") {
      MaxPool<T> pool;
      SegmentOpsKernel<
          T, IndexT, ArrangeHelper<IndexT>,
          MaxPool<T>><<<config.block_per_grid.x, config.thread_per_block.x, 0,
                        ctx.stream()>>>(segment_ids.data<IndexT>(),
                                        input.data<T>(), output->data<T>(), h,
                                        pool);
    } else if (pooltype == "MIN") {
      MinPool<T> pool;
      SegmentOpsKernel<
          T, IndexT, ArrangeHelper<IndexT>,
          MinPool<T>><<<config.block_per_grid.x, config.thread_per_block.x, 0,
                        ctx.stream()>>>(segment_ids.data<IndexT>(),
                                        input.data<T>(), output->data<T>(), h,
                                        pool);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported segment pooling operation, Only MEAN, SUM, MAX, MIN "
          "available, but got %s.",
          pooltype));
    }
  }
};

template <typename T, typename IndexT>
class SegmentPoolGradFunctor<platform::CUDADeviceContext, T, IndexT> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& out_grad,
                  const framework::Tensor& segments, framework::Tensor* in_grad,
                  const framework::Tensor* summed_ids = nullptr,
                  const std::string pooltype = "SUM") {
    if (pooltype == "MAX" || pooltype == "MIN") {
      SegmentPoolCUDAGradFunctor<T, IndexT>(context, input, segments, output,
                                            out_grad, in_grad, pooltype);
    } else if (pooltype == "MEAN") {
      framework::Tensor mean_grad;
      mean_grad.mutable_data<T>(input.dims(), context.GetPlace());
      framework::TensorCopy(out_grad, context.GetPlace(), context, &mean_grad);
      int len = output.dims()[0];
      int dim = output.numel() / len;
      auto config = platform::GetGpuLaunchConfig1D(context, len);
      SimpleDiv<T><<<config.block_per_grid.x, config.thread_per_block.x, 0,
                     context.stream()>>>(mean_grad.data<T>(),
                                         summed_ids->data<T>(), len, dim);
      GPUGather<T, IndexT>(context, mean_grad, segments, in_grad);
    } else if (pooltype == "SUM") {
      GPUGather<T, IndexT>(context, out_grad, segments, in_grad);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported segment pooling operation, Only MEAN, SUM, MAX, MIN "
          "available, but got %s.",
          pooltype));
    }
  }
};

using CUDA = paddle::platform::CUDADeviceContext;
template class SegmentPoolFunctor<CUDA, float, int>;
template class SegmentPoolFunctor<CUDA, float, int64_t>;
template class SegmentPoolFunctor<CUDA, double, int>;
template class SegmentPoolFunctor<CUDA, double, int64_t>;
template class SegmentPoolGradFunctor<CUDA, float, int>;
template class SegmentPoolGradFunctor<CUDA, float, int64_t>;
template class SegmentPoolGradFunctor<CUDA, double, int>;
template class SegmentPoolGradFunctor<CUDA, double, int64_t>;

}  // namespace operators
}  // namespace paddle
