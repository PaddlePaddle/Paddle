#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include <vector>
#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/math_function_impl.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {
namespace math {

using float16 = paddle::platform::float16;

template <>
void gemm<platform::CUDADeviceContext, float16>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float16 alpha, const float16* A, const float16* B, const float16 beta,
    float16* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  // TODO(kexinzhao): add processing code for compute capability < 53 case
  PADDLE_ENFORCE_GE(context.GetComputeCapability(), 53,
                    "cublas fp16 gemm requires GPU compute capability >= 53");

#if CUDA_VERSION >= 8000
  float h_alpha = static_cast<float>(alpha);
  float h_beta = static_cast<float>(beta);

  hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;
#if CUDA_VERSION >= 9000
  if (context.GetComputeCapability() >= 70) {
    PADDLE_ENFORCE(platform::dynload::hipblasSetMathMode(context.cublas_handle(),
                                                        HIPBLAS_TENSOR_OP_MATH));
    algo = CUBLAS_GEMM_DFALT_TENSOR_OP;
  } else {
    PADDLE_ENFORCE(platform::dynload::hipblasSetMathMode(context.cublas_handle(),
                                                        HIPBLAS_DEFAULT_MATH));
  }
#endif  // CUDA_VERSION >= 9000

  // hipblasHgemm does true FP16 computation which is slow for non-Volta
  // GPUs. So use hipblasGemmEx instead which does pesudo FP16 computation:
  // input/output in fp16, computation in fp32, which can also be accelerated
  // using tensor cores in volta GPUs.
  PADDLE_ENFORCE(platform::dynload::hipblasGemmEx(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &h_alpha, B,
      HIP_R_16F, ldb, A, HIP_R_16F, lda, &h_beta, C, HIP_R_16F, N,
      HIP_R_32F, algo));
#else
  // CUDA 7.5 does not support hipblasGemmEx, hence we fall back to use hgemm
  const half h_alpha = static_cast<const half>(alpha);
  const half h_beta = static_cast<const half>(beta);
  const half* h_A = reinterpret_cast<const half*>(A);
  const half* h_B = reinterpret_cast<const half*>(B);
  half* h_C = reinterpret_cast<half*>(C);

  PADDLE_ENFORCE(platform::dynload::hipblasHgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &h_alpha, h_B, ldb,
      h_A, lda, &h_beta, h_C, N));
#endif  // CUDA_VERSION >= 8000
}

template <>
void gemm<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, N));
}

template <>
void gemm<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, N));
}

template <>
void gemm<platform::CUDADeviceContext, float16>(
    const platform::CUDADeviceContext& context, const bool transA,
    const bool transB, const int M, const int N, const int K,
    const float16 alpha, const float16* A, const int lda, const float16* B,
    const int ldb, const float16 beta, float16* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  const half h_alpha = static_cast<const half>(alpha);
  const half h_beta = static_cast<const half>(beta);
  const half* h_A = reinterpret_cast<const half*>(A);
  const half* h_B = reinterpret_cast<const half*>(B);
  half* h_C = reinterpret_cast<half*>(C);

  // TODO(kexinzhao): add processing code for compute capability < 53 case
  PADDLE_ENFORCE_GE(context.GetComputeCapability(), 53,
                    "cublas Hgemm requires GPU compute capability >= 53");
  PADDLE_ENFORCE(platform::dynload::hipblasHgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &h_alpha, h_B, ldb,
      h_A, lda, &h_beta, h_C, ldc));
}

template <>
void gemm<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const bool transA,
    const bool transB, const int M, const int N, const int K, const float alpha,
    const float* A, const int lda, const float* B, const int ldb,
    const float beta, float* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, ldc));
}

template <>
void gemm<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const bool transA,
    const bool transB, const int M, const int N, const int K,
    const double alpha, const double* A, const int lda, const double* B,
    const int ldb, const double beta, double* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, ldc));
}

template <>
void matmul<platform::CUDADeviceContext, float16>(
    const platform::CUDADeviceContext& context,
    const framework::Tensor& matrix_a, bool trans_a,
    const framework::Tensor& matrix_b, bool trans_b, float16 alpha,
    framework::Tensor* matrix_out, float16 beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in CUDAPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::CUDADeviceContext, float16>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<float16>(),
      matrix_b.data<float16>(), beta, matrix_out->data<float16>());
}

template <>
void matmul<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context,
    const framework::Tensor& matrix_a, bool trans_a,
    const framework::Tensor& matrix_b, bool trans_b, float alpha,
    framework::Tensor* matrix_out, float beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in CUDAPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::CUDADeviceContext, float>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<float>(),
      matrix_b.data<float>(), beta, matrix_out->data<float>());
}

template <>
void matmul<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context,
    const framework::Tensor& matrix_a, bool trans_a,
    const framework::Tensor& matrix_b, bool trans_b, double alpha,
    framework::Tensor* matrix_out, double beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in CUDAPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::CUDADeviceContext, double>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<double>(),
      matrix_b.data<double>(), beta, matrix_out->data<double>());
}

template <>
void batched_gemm<platform::CUDADeviceContext, float16>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float16 alpha, const float16* A, const float16* B, const float16 beta,
    float16* C, const int batchCount, const int64_t strideA,
    const int64_t strideB) {
#if CUDA_VERSION >= 8000
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int64_t strideC = M * N;

  const half h_alpha = static_cast<const half>(alpha);
  const half h_beta = static_cast<const half>(beta);
  const half* h_A = reinterpret_cast<const half*>(A);
  const half* h_B = reinterpret_cast<const half*>(B);
  half* h_C = reinterpret_cast<half*>(C);

  // TODO(kexinzhao): add processing code for compute capability < 53 case
  PADDLE_ENFORCE_GE(context.GetComputeCapability(), 53,
                    "cublas Hgemm requires GPU compute capability >= 53");

  PADDLE_ENFORCE(platform::dynload::hipblasHgemmStridedBatched(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &h_alpha, h_B, ldb,
      strideB, h_A, lda, strideA, &h_beta, h_C, ldc, strideC, batchCount));
#else
  PADDLE_ENFORCE(false, "HgemmStridedBatched is not supported on cuda <= 7.5");
#endif
}

template <>
void batched_gemm<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C, const int batchCount, const int64_t strideA,
    const int64_t strideB) {
#if CUDA_VERSION >= 8000
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int64_t strideC = M * N;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemmStridedBatched(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
      strideB, A, lda, strideA, &beta, C, ldc, strideC, batchCount));
#else
  PADDLE_ENFORCE(false, "SgemmStridedBatched is not supported on cuda <= 7.5");
#endif
}

template <>
void batched_gemm<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C, const int batchCount, const int64_t strideA,
    const int64_t strideB) {
#if CUDA_VERSION >= 8000
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int64_t strideC = M * N;

  PADDLE_ENFORCE(platform::dynload::hipblasDgemmStridedBatched(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
      strideB, A, lda, strideA, &beta, C, ldc, strideC, batchCount));
#else
  PADDLE_ENFORCE(false, "DgemmStridedBatched is not supported on cuda <= 7.5");
#endif
}

template <>
void gemv<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const bool trans_a, const int M,
    const int N, const float alpha, const float* A, const float* B,
    const float beta, float* C) {
  hipblasOperation_t cuTransA = (trans_a == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemv(context.cublas_handle(),
                                                cuTransA, N, M, &alpha, A, N, B,
                                                1, &beta, C, 1));
}

template <>
void gemv<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const bool trans_a, const int M,
    const int N, const double alpha, const double* A, const double* B,
    const double beta, double* C) {
  hipblasOperation_t cuTransA = (trans_a == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemv(context.cublas_handle(),
                                                cuTransA, N, M, &alpha, A, N, B,
                                                1, &beta, C, 1));
}

template <>
void axpy<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const int n, const float alpha,
    const float* x, float* y) {
  PADDLE_ENFORCE(platform::dynload::hipblasSaxpy(context.cublas_handle(), n,
                                                &alpha, x, 1, y, 1));
}

template <>
void axpy<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const int n, const double alpha,
    const double* x, double* y) {
  PADDLE_ENFORCE(platform::dynload::hipblasDaxpy(context.cublas_handle(), n,
                                                &alpha, x, 1, y, 1));
}

template struct SetConstant<platform::CUDADeviceContext, platform::float16>;
template struct SetConstant<platform::CUDADeviceContext, float>;
template struct SetConstant<platform::CUDADeviceContext, double>;
template struct SetConstant<platform::CUDADeviceContext, int>;
template struct SetConstant<platform::CUDADeviceContext, int64_t>;
template struct SetConstant<platform::CUDADeviceContext, bool>;

#define DEFINE_GPU_TRANS(RANK)                                         \
  template struct Transpose<platform::CUDADeviceContext, float, RANK>; \
  template struct Transpose<platform::CUDADeviceContext, double, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const platform::DeviceContext& context,
                       framework::Tensor* tensor, float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void operator()() const {
    SetConstant<platform::CUDADeviceContext, T> functor;
    functor(reinterpret_cast<const platform::CUDADeviceContext&>(context_),
            tensor_, static_cast<T>(value_));
  }

  const platform::DeviceContext& context_;
  framework::Tensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<platform::CUDAPlace>(
    const platform::DeviceContext& context, framework::Tensor* tensor,
    float value) {
  framework::VisitDataType(framework::ToDataType(tensor->type()),
                           TensorSetConstantGPU(context, tensor, value));
}

template <typename T>
__global__ void RowwiseAddKernel(const T* a, const T* b, T* c, int width,
                                 int num) {
  T tmp = 1.0 / width;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    int h = i * tmp;
    int w = i - h * width;
    c[i] = a[i] + b[w];
  }
}

template <typename T>
struct RowwiseAdd<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& vector, framework::Tensor* output) {
    auto in_dims = input.dims();
    auto size = input.numel() / in_dims[0];
    PADDLE_ENFORCE_EQ(vector.numel(), size);
    PADDLE_ENFORCE_EQ(output->dims(), in_dims);
    int blocks = 512;
    int grids = (input.numel() + blocks - 1) / blocks;
    RowwiseAddKernel<T><<<grids, blocks, 0, context.stream()>>>(
        input.data<T>(), vector.data<T>(), output->data<T>(),
        static_cast<int>(in_dims[1]), static_cast<int>(input.numel()));
  }
};

template struct RowwiseAdd<platform::CUDADeviceContext, float>;
template struct RowwiseAdd<platform::CUDADeviceContext, double>;
template struct ColwiseSum<platform::CUDADeviceContext, float>;
template struct ColwiseSum<platform::CUDADeviceContext, int>;
template struct ColwiseSum<platform::CUDADeviceContext, int64_t>;
// template struct ColwiseSum<platform::CUDADeviceContext, double>;
// The ColwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), size);
  framework::Tensor one;
  one.mutable_data<double>({in_dims[0]}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  gemv<platform::CUDADeviceContext, double>(
      context, true, static_cast<int>(in_dims[0]), static_cast<int>(in_dims[1]),
      1.0, input.data<double>(), one.data<double>(), 0.0,
      vector->data<double>());
}

template struct RowwiseSum<platform::CUDADeviceContext, float>;
// template struct RowwiseSum<platform::CUDADeviceContext, double>;
// TODO(zcd): Following ColwiseSum format, need to confirm.
// The RowwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void RowwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), in_dims[0]);
  framework::Tensor one;
  one.mutable_data<double>({size}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  gemv<platform::CUDADeviceContext, double>(
      context, true, static_cast<int>(in_dims[1]), static_cast<int>(in_dims[0]),
      1.0, one.data<double>(), input.data<double>(), 0.0,
      vector->data<double>());
}

template struct RowwiseMean<platform::CUDADeviceContext, float>;
template struct RowwiseMean<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
