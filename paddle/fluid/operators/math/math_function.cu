#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <algorithm>
#include <vector>
#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/math_function_impl.h"
#include "paddle/fluid/platform/complex128.h"
#include "paddle/fluid/platform/complex64.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {
namespace math {

using float16 = paddle::platform::float16;
using bfloat16 = paddle::platform::bfloat16;
using complex64 = paddle::platform::complex64;
using complex128 = paddle::platform::complex128;

template struct SetConstant<platform::CUDADeviceContext, platform::float16>;
template struct SetConstant<platform::CUDADeviceContext, float>;
template struct SetConstant<platform::CUDADeviceContext, double>;
template struct SetConstant<platform::CUDADeviceContext, int>;
template struct SetConstant<platform::CUDADeviceContext, int64_t>;
template struct SetConstant<platform::CUDADeviceContext, bool>;
template struct SetConstant<platform::CUDADeviceContext, platform::complex64>;
template struct SetConstant<platform::CUDADeviceContext, platform::complex128>;

#define DEFINE_GPU_TRANS(RANK)                                             \
  template struct Transpose<platform::CUDADeviceContext, float, RANK>;     \
  template struct Transpose<platform::CUDADeviceContext, double, RANK>;    \
  template struct Transpose<platform::CUDADeviceContext, float16, RANK>;   \
  template struct Transpose<platform::CUDADeviceContext, bfloat16, RANK>;  \
  template struct Transpose<platform::CUDADeviceContext, int8_t, RANK>;    \
  template struct Transpose<platform::CUDADeviceContext, int32_t, RANK>;   \
  template struct Transpose<platform::CUDADeviceContext, int64_t, RANK>;   \
  template struct Transpose<platform::CUDADeviceContext, complex64, RANK>; \
  template struct Transpose<platform::CUDADeviceContext, complex128, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

#define REINTERPRET(T, DST_PTR, SRC_PTR) \
  T* DST_PTR = reinterpret_cast<T*>(SRC_PTR)

template <typename T>
__global__ void TransposeNormalKernel(const T* in_ptr, T* out_ptr,
                                      int64_t element,
                                      const int64_t* in_stride_ptr,
                                      const int64_t* out_stride_ptr,
                                      const int64_t* axis_ptr, int rank) {
  CUDA_KERNEL_LOOP(out_idx, element) {
    int64_t in_idx = 0;
    int64_t tmp_idx = out_idx;
    for (int i = 0; i < rank; ++i) {
      const int64_t coordinate = tmp_idx / out_stride_ptr[i];
      tmp_idx -= coordinate * out_stride_ptr[i];
      in_idx += coordinate * in_stride_ptr[axis_ptr[i]];
    }
    out_ptr[out_idx] = in_ptr[in_idx];
  }
}

template <typename T>
struct TransposeNormal<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& in, framework::Tensor* out,
                  const std::vector<int>& axis) {
    const int rank = axis.size();
    auto in_stride = framework::stride(in.dims());
    auto out_stride = framework::stride(out->dims());
    auto* in_ptr = in.data<T>();
    auto* out_ptr = out->data<T>();

    // copy in_stride, out_stride, axis to gpu device
    const platform::CUDAPlace& cuda_place =
        BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace());
    platform::CPUPlace cpu_place = platform::CPUPlace();
    size_t size = 3 * rank * sizeof(int64_t);
    auto cpu_buf_holder = memory::AllocShared(cpu_place, size);
    auto cuda_buf_holder = memory::AllocShared(cuda_place, size);
    REINTERPRET(int64_t, cpu_buf, cpu_buf_holder->ptr());
    REINTERPRET(int64_t, cuda_buf, cuda_buf_holder->ptr());
    for (int i = 0; i < rank; ++i) {
      cpu_buf[i] = in_stride[i];
      cpu_buf[rank + i] = out_stride[i];
      cpu_buf[2 * rank + i] = axis[i];
    }
    memory::Copy(cuda_place, cuda_buf, cpu_place, cpu_buf, size,
                 context.stream());
    REINTERPRET(const int64_t, in_stride_ptr, cuda_buf);
    REINTERPRET(const int64_t, out_stride_ptr, cuda_buf + rank);
    REINTERPRET(const int64_t, axis_ptr, cuda_buf + 2 * rank);

    const int MAX_BLOCK_DIM = context.GetMaxThreadsPerBlock();
    const int MAX_GRID_DIM =
        context.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
    int64_t elements = in.numel();
    int block_size = (elements >= MAX_BLOCK_DIM)
                         ? MAX_BLOCK_DIM
                         : (1 << static_cast<int>(std::log2(elements)));
    int grid_size = elements / block_size;
    grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
    TransposeNormalKernel<T><<<grid_size, block_size, 0, context.stream()>>>(
        in_ptr, out_ptr, elements, in_stride_ptr, out_stride_ptr, axis_ptr,
        rank);
  }
};

// define transpose normal
#define DEFINE_GPU_TRANS_NORMAL(TYPE) \
  template struct TransposeNormal<platform::CUDADeviceContext, TYPE>

DEFINE_GPU_TRANS_NORMAL(float16);
DEFINE_GPU_TRANS_NORMAL(bfloat16);
DEFINE_GPU_TRANS_NORMAL(float);
DEFINE_GPU_TRANS_NORMAL(double);
DEFINE_GPU_TRANS_NORMAL(int);
DEFINE_GPU_TRANS_NORMAL(int64_t);
DEFINE_GPU_TRANS_NORMAL(bool);
DEFINE_GPU_TRANS_NORMAL(int16_t);
DEFINE_GPU_TRANS_NORMAL(uint8_t);
DEFINE_GPU_TRANS_NORMAL(int8_t);
DEFINE_GPU_TRANS_NORMAL(complex64);
DEFINE_GPU_TRANS_NORMAL(complex128);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const platform::DeviceContext& context,
                       framework::Tensor* tensor, float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void apply() const {
    SetConstant<platform::CUDADeviceContext, T> functor;
    functor(reinterpret_cast<const platform::CUDADeviceContext&>(context_),
            tensor_, static_cast<T>(value_));
  }

  const platform::DeviceContext& context_;
  framework::Tensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<platform::CUDAPlace>(
    const platform::DeviceContext& context, framework::Tensor* tensor,
    float value) {
  framework::VisitDataType(tensor->type(),
                           TensorSetConstantGPU(context, tensor, value));
}

template <typename T>
__global__ void RowwiseAddKernel(const T* a, const T* b, T* c, int width,
                                 int num) {
  T tmp = 1.0 / width;
  CUDA_KERNEL_LOOP(i, num) {
    int h = i * tmp;
    int w = i - h * width;
    c[i] = a[i] + b[w];
  }
}

template <typename T>
struct RowwiseAdd<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& vector, framework::Tensor* output) {
    auto in_dims = input.dims();
    auto out_dims = output->dims();
    auto size = input.numel() / in_dims[0];
    PADDLE_ENFORCE_EQ(
        vector.numel(), size,
        platform::errors::InvalidArgument(
            "The input vector size"
            " should be equal to the size of each row of input tensor."
            " Expected vector size=%d, but received %d",
            size, vector.numel()));
    const char* in_dims_cstr = in_dims.to_str().c_str();
    const char* out_dims_cstr = out_dims.to_str().c_str();
    PADDLE_ENFORCE_EQ(
        out_dims, in_dims,
        platform::errors::InvalidArgument(
            "The output tensor shape should be same as the input tensor"
            " shape. Expected output tensor shape: %s,"
            " but received %s",
            in_dims_cstr, out_dims_cstr));
    int blocks = 512;
    int grids = (input.numel() + blocks - 1) / blocks;
    RowwiseAddKernel<T><<<grids, blocks, 0, context.stream()>>>(
        input.data<T>(), vector.data<T>(), output->data<T>(),
        static_cast<int>(in_dims[1]), static_cast<int>(input.numel()));
  }
};

template struct RowwiseAdd<platform::CUDADeviceContext, float>;
template struct RowwiseAdd<platform::CUDADeviceContext, double>;
template struct ColwiseSum<platform::CUDADeviceContext, float>;
template struct ColwiseSum<platform::CUDADeviceContext, int>;
template struct ColwiseSum<platform::CUDADeviceContext, int64_t>;
// template struct ColwiseSum<platform::CUDADeviceContext, double>;
// The ColwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), size,
                    platform::errors::InvalidArgument(
                        "The size of input vector"
                        " should be equal to the size of input tensor column"
                        " dimension. Expected vector size=%d, but received %d",
                        size, vector->numel()));
  framework::Tensor one;
  one.mutable_data<double>({in_dims[0]}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  GetBlas<platform::CUDADeviceContext, double>(context).GEMV(
      true, static_cast<int>(in_dims[0]), static_cast<int>(in_dims[1]), 1.0,
      input.data<double>(), one.data<double>(), 0.0, vector->data<double>());
}

template struct RowwiseSum<platform::CUDADeviceContext, float>;
// template struct RowwiseSum<platform::CUDADeviceContext, double>;
// TODO(zcd): Following ColwiseSum format, need to confirm.
// The RowwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void RowwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), in_dims[0],
                    platform::errors::InvalidArgument(
                        "The size of input vector"
                        " should be equal to the size of input tensor row"
                        " dimension. Expected vector size=%d, but received %d",
                        in_dims[0], vector->numel()));
  framework::Tensor one;
  one.mutable_data<double>({size}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  GetBlas<platform::CUDADeviceContext, double>(context).GEMV(
      true, static_cast<int>(in_dims[1]), static_cast<int>(in_dims[0]), 1.0,
      one.data<double>(), input.data<double>(), 0.0, vector->data<double>());
}

template struct RowwiseMean<platform::CUDADeviceContext, float>;
template struct RowwiseMean<platform::CUDADeviceContext, double>;

template <typename T>
struct ElementwiseAddTo<platform::CUDADeviceContext, T> {
  void operator()(platform::CUDADeviceContext* ctx,
                  const framework::Tensor& src, framework::Tensor* dst) {
    auto in = framework::EigenVector<T>::Flatten(src);
    auto out = framework::EigenVector<T>::Flatten(*dst);
    auto& place = *(ctx->eigen_device());
    out.device(place) = out + in;
  }
};

template struct ElementwiseAddTo<platform::CUDADeviceContext,
                                 platform::float16>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
