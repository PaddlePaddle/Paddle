#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/math_function_impl.h"

namespace paddle {
namespace operators {
namespace math {

template <>
void gemm<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, N));
}

template <>
void gemm<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, N));
}

template <>
void gemm<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const bool transA,
    const bool transB, const int M, const int N, const int K, const float alpha,
    const float* A, const int lda, const float* B, const int ldb,
    const float beta, float* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, ldc));
}

template <>
void gemm<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const bool transA,
    const bool transB, const int M, const int N, const int K,
    const double alpha, const double* A, const int lda, const double* B,
    const int ldb, const double beta, double* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A,
      lda, &beta, C, ldc));
}

template <>
void matmul<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context,
    const framework::Tensor& matrix_a, bool trans_a,
    const framework::Tensor& matrix_b, bool trans_b, float alpha,
    framework::Tensor* matrix_out, float beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in CUDAPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::CUDADeviceContext, float>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<float>(),
      matrix_b.data<float>(), beta, matrix_out->data<float>());
}

template <>
void matmul<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context,
    const framework::Tensor& matrix_a, bool trans_a,
    const framework::Tensor& matrix_b, bool trans_b, double alpha,
    framework::Tensor* matrix_out, double beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in CUDAPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::CUDADeviceContext, double>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<double>(),
      matrix_b.data<double>(), beta, matrix_out->data<double>());
}

template <>
void batched_gemm<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C, const int batchCount, const int strideA, const int strideB) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int strideC = M * N;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemmStridedBatched(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
      strideB, A, lda, strideA, &beta, C, ldc, strideC, batchCount));
}

template <>
void batched_gemm<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C, const int batchCount, const int strideA, const int strideB) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int strideC = M * N;

  PADDLE_ENFORCE(platform::dynload::hipblasDgemmStridedBatched(
      context.cublas_handle(), cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
      strideB, A, lda, strideA, &beta, C, ldc, strideC, batchCount));
}

template <>
void gemv<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const bool trans_a, const int M,
    const int N, const float alpha, const float* A, const float* B,
    const float beta, float* C) {
  hipblasOperation_t cuTransA = (trans_a == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemv(context.cublas_handle(),
                                                cuTransA, N, M, &alpha, A, N, B,
                                                1, &beta, C, 1));
}

template <>
void gemv<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const bool trans_a, const int M,
    const int N, const double alpha, const double* A, const double* B,
    const double beta, double* C) {
  hipblasOperation_t cuTransA = (trans_a == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemv(context.cublas_handle(),
                                                cuTransA, N, M, &alpha, A, N, B,
                                                1, &beta, C, 1));
}

template <>
void axpy<platform::CUDADeviceContext, float>(
    const platform::CUDADeviceContext& context, const int n, const float alpha,
    const float* x, float* y) {
  PADDLE_ENFORCE(platform::dynload::hipblasSaxpy(context.cublas_handle(), n,
                                                &alpha, x, 1, y, 1));
}

template <>
void axpy<platform::CUDADeviceContext, double>(
    const platform::CUDADeviceContext& context, const int n, const double alpha,
    const double* x, double* y) {
  PADDLE_ENFORCE(platform::dynload::hipblasDaxpy(context.cublas_handle(), n,
                                                &alpha, x, 1, y, 1));
}

template struct SetConstant<platform::CUDADeviceContext, float>;
template struct SetConstant<platform::CUDADeviceContext, double>;
template struct SetConstant<platform::CUDADeviceContext, int>;
template struct SetConstant<platform::CUDADeviceContext, int64_t>;
template struct SetConstant<platform::CUDADeviceContext, bool>;

#define DEFINE_GPU_TRANS(RANK)                                         \
  template struct Transpose<platform::CUDADeviceContext, float, RANK>; \
  template struct Transpose<platform::CUDADeviceContext, double, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const platform::DeviceContext& context,
                       framework::Tensor* tensor, float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void operator()() const {
    SetConstant<platform::CUDADeviceContext, T> functor;
    functor(reinterpret_cast<const platform::CUDADeviceContext&>(context_),
            tensor_, static_cast<T>(value_));
  }

  const platform::DeviceContext& context_;
  framework::Tensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<platform::CUDAPlace>(
    const platform::DeviceContext& context, framework::Tensor* tensor,
    float value) {
  framework::VisitDataType(framework::ToDataType(tensor->type()),
                           TensorSetConstantGPU(context, tensor, value));
}

template <typename T>
__global__ void RowwiseAddKernel(const T* a, const T* b, T* c, int width,
                                 int num) {
  T tmp = 1.0 / width;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    int h = i * tmp;
    int w = i - h * width;
    c[i] = a[i] + b[w];
  }
}

template <typename T>
struct RowwiseAdd<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& vector, framework::Tensor* output) {
    auto in_dims = input.dims();
    auto size = input.numel() / in_dims[0];
    PADDLE_ENFORCE_EQ(vector.numel(), size);
    PADDLE_ENFORCE_EQ(output->dims(), in_dims);
    int blocks = 512;
    int grids = (input.numel() + blocks - 1) / blocks;
    RowwiseAddKernel<T><<<grids, blocks, 0, context.stream()>>>(
        input.data<T>(), vector.data<T>(), output->data<T>(),
        static_cast<int>(in_dims[1]), static_cast<int>(input.numel()));
  }
};

template struct RowwiseAdd<platform::CUDADeviceContext, float>;
template struct RowwiseAdd<platform::CUDADeviceContext, double>;
template struct ColwiseSum<platform::CUDADeviceContext, float>;
// template struct ColwiseSum<platform::CUDADeviceContext, double>;
// The ColwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), size);
  framework::Tensor one;
  one.mutable_data<double>({in_dims[0]}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  gemv<platform::CUDADeviceContext, double>(
      context, true, static_cast<int>(in_dims[0]), static_cast<int>(in_dims[1]),
      1.0, input.data<double>(), one.data<double>(), 0.0,
      vector->data<double>());
}

template struct RowwiseSum<platform::CUDADeviceContext, float>;
// template struct RowwiseSum<platform::CUDADeviceContext, double>;
// TODO(zcd): Following ColwiseSum format, need to confirm.
// The RowwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void RowwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), in_dims[0]);
  framework::Tensor one;
  one.mutable_data<double>({size}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  gemv<platform::CUDADeviceContext, double>(
      context, true, static_cast<int>(in_dims[1]), static_cast<int>(in_dims[0]),
      1.0, one.data<double>(), input.data<double>(), 0.0,
      vector->data<double>());
}

template struct RowwiseMean<platform::CUDADeviceContext, float>;
template struct RowwiseMean<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
