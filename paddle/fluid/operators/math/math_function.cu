#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <vector>
#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/math_function_impl.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {
namespace math {

using float16 = paddle::platform::float16;

template struct SetConstant<platform::CUDADeviceContext, platform::float16>;
template struct SetConstant<platform::CUDADeviceContext, float>;
template struct SetConstant<platform::CUDADeviceContext, double>;
template struct SetConstant<platform::CUDADeviceContext, int>;
template struct SetConstant<platform::CUDADeviceContext, int64_t>;
template struct SetConstant<platform::CUDADeviceContext, bool>;

#define DEFINE_GPU_TRANS(RANK)                                           \
  template struct Transpose<platform::CUDADeviceContext, float, RANK>;   \
  template struct Transpose<platform::CUDADeviceContext, double, RANK>;  \
  template struct Transpose<platform::CUDADeviceContext, float16, RANK>; \
  template struct Transpose<platform::CUDADeviceContext, int8_t, RANK>;  \
  template struct Transpose<platform::CUDADeviceContext, int32_t, RANK>; \
  template struct Transpose<platform::CUDADeviceContext, int64_t, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const platform::DeviceContext& context,
                       framework::Tensor* tensor, float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void apply() const {
    SetConstant<platform::CUDADeviceContext, T> functor;
    functor(reinterpret_cast<const platform::CUDADeviceContext&>(context_),
            tensor_, static_cast<T>(value_));
  }

  const platform::DeviceContext& context_;
  framework::Tensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<platform::CUDAPlace>(
    const platform::DeviceContext& context, framework::Tensor* tensor,
    float value) {
  framework::VisitDataType(tensor->type(),
                           TensorSetConstantGPU(context, tensor, value));
}

template <typename T>
__global__ void RowwiseAddKernel(const T* a, const T* b, T* c, int width,
                                 int num) {
  T tmp = 1.0 / width;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    int h = i * tmp;
    int w = i - h * width;
    c[i] = a[i] + b[w];
  }
}

template <typename T>
struct RowwiseAdd<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& vector, framework::Tensor* output) {
    auto in_dims = input.dims();
    auto size = input.numel() / in_dims[0];
    PADDLE_ENFORCE_EQ(vector.numel(), size);
    PADDLE_ENFORCE_EQ(output->dims(), in_dims);
    int blocks = 512;
    int grids = (input.numel() + blocks - 1) / blocks;
    RowwiseAddKernel<T><<<grids, blocks, 0, context.stream()>>>(
        input.data<T>(), vector.data<T>(), output->data<T>(),
        static_cast<int>(in_dims[1]), static_cast<int>(input.numel()));
  }
};

template struct RowwiseAdd<platform::CUDADeviceContext, float>;
template struct RowwiseAdd<platform::CUDADeviceContext, double>;
template struct ColwiseSum<platform::CUDADeviceContext, float>;
template struct ColwiseSum<platform::CUDADeviceContext, int>;
template struct ColwiseSum<platform::CUDADeviceContext, int64_t>;
// template struct ColwiseSum<platform::CUDADeviceContext, double>;
// The ColwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), size);
  framework::Tensor one;
  one.mutable_data<double>({in_dims[0]}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  GetBlas<platform::CUDADeviceContext, double>(context).GEMV(
      true, static_cast<int>(in_dims[0]), static_cast<int>(in_dims[1]), 1.0,
      input.data<double>(), one.data<double>(), 0.0, vector->data<double>());
}

template struct RowwiseSum<platform::CUDADeviceContext, float>;
// template struct RowwiseSum<platform::CUDADeviceContext, double>;
// TODO(zcd): Following ColwiseSum format, need to confirm.
// The RowwiseSum<platform::CUDADeviceContext, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void RowwiseSum<platform::CUDADeviceContext, double>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), in_dims[0]);
  framework::Tensor one;
  one.mutable_data<double>({size}, context.GetPlace());
  SetConstant<platform::CUDADeviceContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  GetBlas<platform::CUDADeviceContext, double>(context).GEMV(
      true, static_cast<int>(in_dims[1]), static_cast<int>(in_dims[0]), 1.0,
      one.data<double>(), input.data<double>(), 0.0, vector->data<double>());
}

template struct RowwiseMean<platform::CUDADeviceContext, float>;
template struct RowwiseMean<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
