#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/sequence_pooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

#define FLT_MAX __FLT_MAX__

template <typename T>
struct MaxPoolFunctor {
  HOSTDEVICE void operator()(const T* input, const size_t start,
                             const size_t end, const size_t item_dim, T* output,
                             int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      T max_val = static_cast<T>(-FLT_MAX);
      int max_index = -1;
      for (int i = start; i < end; ++i) {
        if (max_val < input[item_dim * i + tid]) {
          max_val = input[item_dim * i + tid];
          max_index = i;
        }
      }
      output[tid] = max_val;
      index[tid] = max_index;
    }
  }
};

template <typename T>
struct AvgPoolFunctor {
  HOSTDEVICE void operator()(const T* input, const size_t start,
                             const size_t end, const size_t item_dim, T* output,
                             int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      T val = static_cast<T>(0);
      for (int i = start; i < end; ++i) {
        val += input[item_dim * i + tid];
      }
      // end, start is lod, so end - start != 0
      output[tid] = val / static_cast<T>(end - start);
    }
  }
};

template <typename T>
struct SumPoolFunctor {
  HOSTDEVICE void operator()(const T* input, const size_t start,
                             const size_t end, const size_t item_dim, T* output,
                             int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      T val = static_cast<T>(0);
      for (int i = start; i < end; ++i) {
        val += input[item_dim * i + tid];
      }
      output[tid] = val;
    }
  }
};

template <typename T>
struct SqrtPoolFunctor {
  HOSTDEVICE void operator()(const T* input, const size_t start,
                             const size_t end, const size_t item_dim, T* output,
                             int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      T val = static_cast<T>(0);
      for (int i = start; i < end; ++i) {
        val += input[item_dim * i + tid];
      }
      // end, start is lod, so end - start != 0
      output[tid] = val / sqrt(end - start);
    }
  }
};

template <typename T>
struct LastPoolFunctor {
  HOSTDEVICE void operator()(const T* input, const size_t start,
                             const size_t end, const size_t item_dim, T* output,
                             int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      output[tid] = input[item_dim * (end - 1) + tid];
    }
  }
};

template <typename T>
struct FirstPoolFunctor {
  HOSTDEVICE void operator()(const T* input, const size_t start,
                             const size_t end, const size_t item_dim, T* output,
                             int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      output[tid] = input[item_dim * start + tid];
    }
  }
};

template <typename T, typename Range_OP>
__global__ void sequence_pool_kernel(Range_OP op, const T* input,
                                     const size_t* lod, const size_t lod_size,
                                     const size_t item_dim, T* output,
                                     int* index) {
  int bid = blockIdx.x;
  if (bid >= lod_size - 1) return;
  size_t start = lod[bid];
  size_t end = lod[bid + 1];
  int* index_offset = nullptr;
  if (index != nullptr) {
    index_offset = &index[bid * item_dim];
  }
  op(input, start, end, item_dim, &output[bid * item_dim], index_offset);
}

template <typename T>
class SequencePoolFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const std::string pooltype, const framework::LoDTensor& input,
                  framework::Tensor* output, bool is_test,
                  framework::Tensor* index = nullptr) {
    auto& lod = input.lod()[0];
    const size_t item_dim = output->numel() / output->dims()[0];
    dim3 threads(1024, 1);
    dim3 grid(lod.size(), 1);
    if (pooltype == "MAX") {
      sequence_pool_kernel<
          T, MaxPoolFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          MaxPoolFunctor<T>(), input.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          output->mutable_data<T>(context.GetPlace()), index->data<int>());
    } else if (pooltype == "AVERAGE") {
      sequence_pool_kernel<
          T, AvgPoolFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          AvgPoolFunctor<T>(), input.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          output->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "SUM") {
      sequence_pool_kernel<
          T, SumPoolFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          SumPoolFunctor<T>(), input.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          output->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "SQRT") {
      sequence_pool_kernel<
          T, SqrtPoolFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          SqrtPoolFunctor<T>(), input.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          output->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "LAST") {
      sequence_pool_kernel<
          T, LastPoolFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          LastPoolFunctor<T>(), input.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          output->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "FIRST") {
      sequence_pool_kernel<
          T, FirstPoolFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          FirstPoolFunctor<T>(), input.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          output->mutable_data<T>(context.GetPlace()), nullptr);
    } else {
      PADDLE_THROW("unsupported pooling pooltype");
    }
  }
};

template <typename T>
struct MaxPoolGradFunctor {
  HOSTDEVICE void operator()(const T* out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T* in_grad, const int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        if (i == index[tid]) {
          in_grad[item_dim * i + tid] = out_grad[tid];
        } else {
          in_grad[item_dim * i + tid] = static_cast<T>(0);
        }
      }
    }
  }
};

template <typename T>
struct AvgPoolGradFunctor {
  HOSTDEVICE void operator()(const T* out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T* in_grad, const int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        in_grad[item_dim * i + tid] = out_grad[tid] / (end - start);
      }
    }
  }
};

template <typename T>
struct SumPoolGradFunctor {
  HOSTDEVICE void operator()(const T* out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T* in_grad, const int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        in_grad[item_dim * i + tid] = out_grad[tid];
      }
    }
  }
};

template <typename T>
struct SqrtPoolGradFunctor {
  HOSTDEVICE void operator()(const T* out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T* in_grad, const int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        in_grad[item_dim * i + tid] =
            out_grad[tid] / (sqrt(static_cast<T>(end - start)));
      }
    }
  }
};

template <typename T>
struct LastPoolGradFunctor {
  HOSTDEVICE void operator()(const T* out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T* in_grad, const int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        if (i == end - 1) {
          in_grad[item_dim * i + tid] = out_grad[tid];
        } else {
          in_grad[item_dim * i + tid] = static_cast<T>(0);
        }
      }
    }
  }
};

template <typename T>
struct FirstPoolGradFunctor {
  HOSTDEVICE void operator()(const T* out_grad, const size_t start,
                             const size_t end, const size_t item_dim,
                             T* in_grad, const int* index) {
    for (int tid = threadIdx.x; tid < item_dim; tid += blockDim.x) {
      for (int i = start; i < end; ++i) {
        if (i == start) {
          in_grad[item_dim * i + tid] = out_grad[tid];
        } else {
          in_grad[item_dim * i + tid] = static_cast<T>(0);
        }
      }
    }
  }
};

template <typename T, typename Range_OP>
__global__ void sequence_pool_grad_kernel(Range_OP op, const T* out_grad,
                                          const size_t* lod,
                                          const size_t lod_size,
                                          const size_t item_dim, T* in_grad,
                                          const int* index) {
  int bid = blockIdx.x;
  if (bid >= lod_size - 1) return;
  size_t start = lod[bid];
  size_t end = lod[bid + 1];
  const int* index_offset = nullptr;
  if (index != nullptr) {
    index_offset = &index[bid * item_dim];
  }
  op(&out_grad[bid * item_dim], start, end, item_dim, in_grad, index_offset);
}

template <typename T>
class SequencePoolGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const std::string pooltype, const framework::Tensor& out_grad,
                  framework::LoDTensor* in_grad,
                  /* max pool has index */
                  const framework::Tensor* index = nullptr) {
    auto& lod = in_grad->lod()[0];
    const size_t item_dim = in_grad->numel() / in_grad->dims()[0];
    dim3 threads(1024, 1);
    dim3 grid(lod.size(), 1);
    if (pooltype == "MAX") {
      sequence_pool_grad_kernel<
          T, MaxPoolGradFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          MaxPoolGradFunctor<T>(), out_grad.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          in_grad->mutable_data<T>(context.GetPlace()), index->data<int>());
    } else if (pooltype == "AVERAGE") {
      sequence_pool_grad_kernel<
          T, AvgPoolGradFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          AvgPoolGradFunctor<T>(), out_grad.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          in_grad->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "SUM") {
      sequence_pool_grad_kernel<
          T, SumPoolGradFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          SumPoolGradFunctor<T>(), out_grad.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          in_grad->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "SQRT") {
      sequence_pool_grad_kernel<
          T, SqrtPoolGradFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          SqrtPoolGradFunctor<T>(), out_grad.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          in_grad->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "LAST") {
      sequence_pool_grad_kernel<
          T, LastPoolGradFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          LastPoolGradFunctor<T>(), out_grad.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          in_grad->mutable_data<T>(context.GetPlace()), nullptr);
    } else if (pooltype == "FIRST") {
      sequence_pool_grad_kernel<
          T, FirstPoolGradFunctor<T>><<<grid, threads, 0, context.stream()>>>(
          FirstPoolGradFunctor<T>(), out_grad.data<T>(),
          lod.CUDAData(context.GetPlace()), lod.size(), item_dim,
          in_grad->mutable_data<T>(context.GetPlace()), nullptr);

    } else {
      PADDLE_THROW("unsupported pooling pooltype");
    }
  }
};

// sequence pooling
template class SequencePoolFunctor<platform::CUDADeviceContext, float>;
template class SequencePoolFunctor<platform::CUDADeviceContext, double>;
template class SequencePoolGradFunctor<platform::CUDADeviceContext, float>;
template class SequencePoolGradFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
