#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/operators/math/sequence_padding.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T, CopyType Type>
__global__ void SequencePaddingKernel(
    T* dst, const T* src, const T* pad_value, bool is_constant_pad,
    const size_t* seq_offsets, const size_t seq_num, const size_t pad_seq_len,
    const size_t step_width, bool norm_by_len, const PadLayout layout) {
  size_t seq_idx = blockIdx.y;
  size_t seq_len = seq_offsets[seq_idx + 1] - seq_offsets[seq_idx];

  size_t step_idx = blockIdx.x * blockDim.y + threadIdx.y;
  size_t seq_data_offset = (seq_offsets[seq_idx] + step_idx) * step_width;
  size_t pad_data_offset = layout == kBatchLengthWidth
                               ? (seq_idx * pad_seq_len + step_idx) * step_width
                               : (step_idx * seq_num + seq_idx) * step_width;

  T* dst_data = dst + (Type == kSeqToPad ? pad_data_offset : seq_data_offset);
  const T* src_data =
      src + (Type == kSeqToPad ? seq_data_offset : pad_data_offset);

  if (step_idx < seq_len) {
    float scale = norm_by_len ? (1.0f / static_cast<float>(seq_len)) : 1.0f;
    for (size_t i = threadIdx.x; i < step_width; i += blockDim.x) {
      dst_data[i] = scale * src_data[i];
    }
  } else if (step_idx < pad_seq_len && Type == kSeqToPad) {
    for (size_t i = threadIdx.x; i < step_width; i += blockDim.x) {
      dst_data[i] = is_constant_pad ? pad_value[0] : pad_value[i];
    }
  }
}

template <typename T>
class PaddingLoDTensorFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::LoDTensor& seq_tensor,
                  framework::LoDTensor* pad_tensor,
                  const framework::LoDTensor& pad_value, int pad_seq_len = -1,
                  int lod_level = 0, bool norm_by_times = false,
                  const PadLayout layout = kBatchLengthWidth) {
    auto seq_lod = seq_tensor.lod();
    const auto seq_offsets = framework::ToAbsOffset(seq_lod)[lod_level];
    const auto& seq_tensor_dims = seq_tensor.dims();
    const auto& pad_tensor_dims = pad_tensor->dims();
    int max_seq_len = MaximumSequenceLength(seq_offsets);
    if (pad_seq_len == -1) {
      pad_seq_len = max_seq_len;
    }
    PADDLE_ENFORCE_GE(pad_seq_len, max_seq_len,
                      "The pad_seq_len must be equal to or greater than the "
                      "original max sequence length.");
    int step_width = seq_tensor.numel() / seq_tensor_dims[0];
    int seq_num = seq_offsets.size() - 1;

    CheckDims(seq_tensor_dims, pad_tensor_dims, seq_offsets, pad_seq_len,
              step_width, layout);
    PADDLE_ENFORCE(pad_value.numel() == 1 || pad_value.numel() == step_width,
                   "The numel of 'pad_value' can only be 1 or be equal to the "
                   "'step_width'.");

    const int kBlockSize = 512;

    /* At least use 32 threads to copy sequence_width elements,
     * and at least 8 elements for each thread.
     */
    size_t block_dim_x =
        std::min(((((step_width + 7) >> 3) + 31) >> 5) << 5, kBlockSize);
    size_t block_dim_y = kBlockSize / block_dim_x;
    dim3 threads(block_dim_x, block_dim_y);

    size_t grid_dim_x = (pad_seq_len + block_dim_y - 1) / block_dim_y;
    size_t grid_dim_y = seq_num;
    dim3 grid(grid_dim_x, grid_dim_y);

    const T* seq_data = seq_tensor.data<T>();
    T* pad_data = pad_tensor->data<T>();
    const T* pad_value_data = pad_value.data<T>();

    SequencePaddingKernel<T, kSeqToPad><<<grid, threads, 0, context.stream()>>>(
        pad_data, seq_data, pad_value_data, pad_value.numel() == 1,
        seq_offsets.CUDAData(context.GetPlace()), seq_num, pad_seq_len,
        step_width, norm_by_times, layout);
  }
};

template <typename T>
class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::LoDTensor& pad_tensor,
                  framework::LoDTensor* seq_tensor, int pad_seq_len = -1,
                  int lod_level = 0, bool norm_by_times = false,
                  const PadLayout layout = kBatchLengthWidth) {
    auto seq_offsets = framework::ToAbsOffset(seq_tensor->lod())[lod_level];
    const auto& seq_tensor_dims = seq_tensor->dims();
    const auto& pad_tensor_dims = pad_tensor.dims();
    int max_seq_len = MaximumSequenceLength(seq_offsets);
    if (pad_seq_len == -1) {
      pad_seq_len = max_seq_len;
    }
    int step_width = seq_tensor->numel() / seq_tensor_dims[0];
    int seq_num = seq_offsets.size() - 1;

    CheckDims(seq_tensor_dims, pad_tensor_dims, seq_offsets, pad_seq_len,
              step_width, layout);
    /*
    if (!norm_by_times && seq_num == 1UL && pad_seq_len == max_seq_len) {
      TensorCopy(pad_tensor, context.GetPlace(), context, seq_tensor);
      seq_tensor->Resize(seq_tensor_dims);
      return;
    }
    */

    const int kBlockSize = 512;

    /* At least use 32 threads to copy sequence_width elements,
     * and at least 8 elements for each thread.
     */
    size_t block_dim_x =
        std::min(((((step_width + 7) >> 3) + 31) >> 5) << 5, kBlockSize);
    size_t block_dim_y = kBlockSize / block_dim_x;
    dim3 threads(block_dim_x, block_dim_y);

    size_t grid_dim_x = (pad_seq_len + block_dim_y - 1) / block_dim_y;
    size_t grid_dim_y = seq_num;
    dim3 grid(grid_dim_x, grid_dim_y);

    const T* pad_data = pad_tensor.data<T>();
    T* seq_data = seq_tensor->data<T>();

    SequencePaddingKernel<T, kPadToSeq><<<grid, threads, 0, context.stream()>>>(
        seq_data, pad_data, nullptr, false,
        seq_offsets.CUDAData(context.GetPlace()), seq_num, pad_seq_len,
        step_width, norm_by_times, layout);
  }
};

template class PaddingLoDTensorFunctor<platform::CUDADeviceContext, int>;
template class PaddingLoDTensorFunctor<platform::CUDADeviceContext, int64_t>;
template class PaddingLoDTensorFunctor<platform::CUDADeviceContext, float>;
template class PaddingLoDTensorFunctor<platform::CUDADeviceContext, double>;

template class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, int>;
template class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, int64_t>;
template class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, float>;
template class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
