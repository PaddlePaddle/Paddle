#include "hip/hip_runtime.h"
/* Copyright (c) 2018 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/operators/math/concat.h"
#include "paddle/fluid/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__device__ T upper_bound(const T* first, T count, T val) {
  const T* orig = first;
  const T* it = nullptr;
  T step = 0;
  while (count > 0) {
    it = first;
    step = count / 2;
    it += step;
    if (!(val < *it)) {
      first = ++it;
      count -= step + 1;
    } else {
      count = step;
    }
  }
  return first - orig;
}

template <typename T>
__global__ void KernelConcat(T** inputs, const int* input_cols, int col_size,
                             const int output_rows, const int output_cols,
                             T* output) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int segment = upper_bound<int>(input_cols, col_size, tid_x) - 1;

  int curr_offset = input_cols[segment];
  int curr_segment = segment;
  for (; tid_x < output_cols; tid_x += blockDim.x * gridDim.x) {
    T curr_col_offset;
    while ((curr_col_offset = input_cols[curr_segment + 1]) <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;
    T* input_ptr = inputs[curr_segment];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < output_rows; tid_y += blockDim.y * gridDim.y)
      output[tid_y * output_cols + tid_x] =
          input_ptr[tid_y * segment_width + local_col];
  }
}

template <typename T>
__global__ void KernelConcat(T** inputs, const int input_col,
                             const int output_rows, const int output_cols,
                             T* output) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  double inv_input_col = 1.0 / input_col;
  for (; tid_x < output_cols; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x * inv_input_col;
    int in_offset = tid_x - split * input_col;
    T* input_ptr = inputs[split];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < output_rows; tid_y += blockDim.y * gridDim.y) {
      output[tid_y * output_cols + tid_x] =
          input_ptr[tid_y * input_col + in_offset];
    }
  }
}

template <typename T>
__global__ void KernelConcatGrad(const T* input, const int input_row,
                                 const int input_col, const int* output_cols,
                                 int col_size, T** outputs) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int segment = upper_bound<int>(output_cols, col_size, tid_x) - 1;
  int curr_offset = output_cols[segment];
  int curr_segment = segment;
  for (; tid_x < input_col; tid_x += blockDim.x * gridDim.x) {
    T curr_col_offset;
    while ((curr_col_offset = output_cols[curr_segment + 1]) <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;
    T* output_ptr = outputs[curr_segment];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < input_row; tid_y += blockDim.y * gridDim.y)
      output_ptr[tid_y * segment_width + local_col] =
          input[tid_y * input_col + tid_x];
  }
}

template <typename T>
__global__ void KernelConcatGrad(const T* input, const int input_row,
                                 const int input_col, const int output_cols,
                                 T** outputs) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  double inv_input_col = 1.0 / input_col;
  for (; tid_x < input_col; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x * inv_input_col;
    int in_offset = tid_x - split * input_col;
    T* output_ptr = outputs[split];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < input_row; tid_y += blockDim.y * gridDim.y)
      output_ptr[tid_y * output_cols + in_offset] =
          input[tid_y * input_col + tid_x];
  }
}

/*
 * All tensors' dimension should be the same and the values of
 * each dimension are the same, except the axis dimension.
 */
template <typename T>
class ConcatFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const std::vector<framework::Tensor>& input, const int axis,
                  framework::Tensor* output) {
    // TODO(zcd): Add input data validity checking
    int num = input.size();
    int rows = 1;
    auto dim_0 = input[0].dims();
    for (int i = 0; i < axis; ++i) {
      rows *= dim_0[i];
    }
    int cols = input[0].numel() / rows;
    int out_rows = rows, out_cols = 0;

    framework::Vector<int16_t> inputs_data(num * sizeof(T*) / 2);
    framework::Vector<int> inputs_cols(num + 1);
    inputs_cols[0] = 0;
    T** inputs_ptr = reinterpret_cast<T**>(inputs_data.data());

    bool sameShape = true;
    for (int i = 0; i < num; ++i) {
      int t_cols = input[i].numel() / rows;
      if (sameShape) {
        if (t_cols != cols) sameShape = false;
      }
      out_cols += t_cols;
      inputs_cols[i + 1] = out_cols;
      inputs_ptr[i] = const_cast<T*>(input[i].data<T>());
    }

    T** ins_gpu =
        reinterpret_cast<T**>(inputs_data.CUDAMutableData(context.GetPlace()));
    const int* ins_col_gpu = inputs_cols.CUDAData(context.GetPlace());

    // computation
    // set the thread block and grid according to CurrentDeviceId
    const int kThreadsPerBlock = 1024;
    int block_cols = kThreadsPerBlock;
    if (out_cols < kThreadsPerBlock) {  // block_cols is aligned by 32.
      block_cols = ((out_cols + 31) >> 5) << 5;
    }
    int block_rows = kThreadsPerBlock / block_cols;
    dim3 block_size = dim3(block_cols, block_rows, 1);

    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    int grid_cols =
        std::min((out_cols + block_cols - 1) / block_cols, max_blocks);
    int grid_rows =
        std::min(max_blocks / grid_cols, std::max(out_rows / block_rows, 1));
    dim3 grid_size = dim3(grid_cols, grid_rows, 1);

    if (sameShape) {
      KernelConcat<<<grid_size, block_size, 0, context.stream()>>>(
          ins_gpu, cols, out_rows, out_cols, output->data<T>());
    } else {
      KernelConcat<<<grid_size, block_size, 0, context.stream()>>>(
          ins_gpu, ins_col_gpu, static_cast<int>(inputs_cols.size()), out_rows,
          out_cols, output->data<T>());
    }
  }
};

/*
 * All tensors' dimension should be the same and the values of
 * each dimension are the same, except the axis dimension.
 */
template <typename T>
class ConcatGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const int axis,
                  std::vector<framework::Tensor>& outputs) {
    // TODO(zcd): Add input data validity checking
    int num = outputs.size();
    int input_row = 1;
    auto dim_0 = outputs[0].dims();
    for (int i = 0; i < axis; ++i) {
      input_row *= dim_0[i];
    }

    int output_col_0 = outputs[0].numel() / input_row;
    int input_col = 0;
    bool sameShape = true;

    framework::Vector<int16_t> outputs_data(num * sizeof(T*) / 2);
    framework::Vector<int> outputs_cols(num + 1);
    outputs_cols[0] = 0;
    T** outputs_ptr = reinterpret_cast<T**>(outputs_data.data());

    for (int i = 0; i < num; ++i) {
      int t_col = outputs[i].numel() / input_row;
      if (sameShape) {
        if (t_col != output_col_0) sameShape = false;
      }
      input_col += t_col;
      outputs_cols[i + 1] = input_col;
      outputs_ptr[i] = outputs[i].data<T>();
    }

    T** outs_gpu =
        reinterpret_cast<T**>(outputs_data.CUDAMutableData(context.GetPlace()));
    const int* outs_col_gpu = outputs_cols.CUDAData(context.GetPlace());

    // computation
    const int kThreadsPerBlock = 1024;
    int block_cols = kThreadsPerBlock;
    if (input_col < kThreadsPerBlock) {  // block_cols is aligned by 32.
      block_cols = ((input_col + 31) >> 5) << 5;
    }
    int block_rows = kThreadsPerBlock / block_cols;
    dim3 block_size = dim3(block_cols, block_rows, 1);

    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    int grid_cols =
        std::min((input_col + block_cols - 1) / block_cols, max_blocks);
    int grid_rows =
        std::min(max_blocks / grid_cols, std::max(input_row / block_rows, 1));
    dim3 grid_size = dim3(grid_cols, grid_rows, 1);

    if (sameShape) {
      KernelConcatGrad<<<grid_size, block_size, 0, context.stream()>>>(
          input.data<T>(), input_row, input_col, output_col_0, outs_gpu);
    } else {
      KernelConcatGrad<<<grid_size, block_size, 0, context.stream()>>>(
          input.data<T>(), input_row, input_col, outs_col_gpu,
          static_cast<int>(outputs_cols.size()), outs_gpu);
    }
  }
};

template class ConcatFunctor<platform::CUDADeviceContext, int>;
template class ConcatFunctor<platform::CUDADeviceContext, int64_t>;
template class ConcatFunctor<platform::CUDADeviceContext, float>;
template class ConcatFunctor<platform::CUDADeviceContext, double>;

template class ConcatGradFunctor<platform::CUDADeviceContext, int>;
template class ConcatGradFunctor<platform::CUDADeviceContext, int64_t>;
template class ConcatGradFunctor<platform::CUDADeviceContext, float>;
template class ConcatGradFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
