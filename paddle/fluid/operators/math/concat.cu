#include "hip/hip_runtime.h"
/* Copyright (c) 2018 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/operators/math/concat.h"
#include "paddle/fluid/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__device__ T upper_bound(const T* first, T count, T val) {
  const T* orig = first;
  const T* it = nullptr;
  T step = 0;
  while (count > 0) {
    it = first;
    step = count / 2;
    it += step;
    if (!(val < *it)) {
      first = ++it;
      count -= step + 1;
    } else {
      count = step;
    }
  }
  return first - orig;
}

template <typename T>
__global__ void KernelConcat(T** inputs, const int* input_cols, int col_size,
                             const int output_rows, const int output_cols,
                             T* output) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int segment = upper_bound<int>(input_cols, col_size, tid_x) - 1;

  int curr_offset = input_cols[segment];
  int curr_segment = segment;
  for (; tid_x < output_cols; tid_x += blockDim.x * gridDim.x) {
    T curr_col_offset;
    while ((curr_col_offset = input_cols[curr_segment + 1]) <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;
    T* input_ptr = inputs[curr_segment];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < output_rows; tid_y += blockDim.y * gridDim.y)
      output[tid_y * output_cols + tid_x] =
          input_ptr[tid_y * segment_width + local_col];
  }
}

template <typename T>
__global__ void KernelConcat(T** inputs_data, const int fixed_in_col,
                             const int out_rows, const int out_cols,
                             T* output_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  for (; tid_x < out_cols; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x * 1.0 / fixed_in_col;
    int in_offset = tid_x - split * fixed_in_col;
    T* input_ptr = inputs_data[split];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < out_rows; tid_y += blockDim.y * gridDim.y) {
      output_data[tid_y * out_cols + tid_x] =
          input_ptr[tid_y * fixed_in_col + in_offset];
    }
  }
}

template <typename T>
__global__ void KernelConcatGrad(const T* input_data, const int in_row,
                                 const int in_col, const int* out_cols,
                                 int out_cols_size, T** outputs_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int segment = upper_bound<int>(out_cols, out_cols_size, tid_x) - 1;
  int curr_offset = out_cols[segment];
  int curr_segment = segment;
  for (; tid_x < in_col; tid_x += blockDim.x * gridDim.x) {
    T curr_col_offset;
    while ((curr_col_offset = out_cols[curr_segment + 1]) <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;
    T* output_ptr = outputs_data[curr_segment];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
      output_ptr[tid_y * segment_width + local_col] =
          input_data[tid_y * in_col + tid_x];
  }
}

template <typename T>
__global__ void KernelConcatGrad(const T* input_data, const int in_row,
                                 const int in_col, const int fixed_out_col,
                                 T** outputs_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  for (; tid_x < in_col; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x / fixed_out_col;
    int in_offset = tid_x - split * fixed_out_col;
    T* output_ptr = outputs_data[split];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
      output_ptr[tid_y * fixed_out_col + in_offset] =
          input_data[tid_y * in_col + tid_x];
  }
}

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */
template <typename T>
class ConcatFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const std::vector<framework::Tensor>& input, const int axis,
                  framework::Tensor* output) {
    // TODO(zcd): Add input data validity checking
    int in_num = input.size();
    int in_row = 1;
    auto dim_0 = input[0].dims();
    for (int i = 0; i < axis; ++i) {
      in_row *= dim_0[i];
    }
    int in_col = input[0].numel() / in_row;
    int out_row = in_row, out_col = 0;

    framework::Vector<int16_t> inputs_data(in_num * sizeof(T*) / 2);
    framework::Vector<int> inputs_col(in_num + 1);
    T** inputs_ptr = reinterpret_cast<T**>(inputs_data.data());

    inputs_col[0] = 0;
    bool sameShape = true;
    for (int i = 0; i < in_num; ++i) {
      int t_cols = input[i].numel() / in_row;
      if (sameShape) {
        if (t_cols != in_col) sameShape = false;
      }
      out_col += t_cols;
      inputs_col[i + 1] = out_col;
      inputs_ptr[i] = const_cast<T*>(input[i].data<T>());
    }

    T** dev_ins_data =
        reinterpret_cast<T**>(inputs_data.CUDAMutableData(context.GetPlace()));

    // computation
    // set the thread block and grid according to CurrentDeviceId
    const int kThreadsPerBlock = 1024;
    int block_cols = kThreadsPerBlock;
    if (out_col < kThreadsPerBlock) {  // block_cols is aligned by 32.
      block_cols = ((out_col + 31) >> 5) << 5;
    }
    int block_rows = kThreadsPerBlock / block_cols;
    dim3 block_size = dim3(block_cols, block_rows, 1);

    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    int grid_cols =
        std::min((out_col + block_cols - 1) / block_cols, max_blocks);
    int grid_rows =
        std::min(max_blocks / grid_cols, std::max(out_row / block_rows, 1));
    dim3 grid_size = dim3(grid_cols, grid_rows, 1);

    if (sameShape) {
      KernelConcat<<<grid_size, block_size, 0, context.stream()>>>(
          dev_ins_data, in_col, out_row, out_col, output->data<T>());
    } else {
      const int* dev_ins_col_data = inputs_col.CUDAData(context.GetPlace());
      KernelConcat<<<grid_size, block_size, 0, context.stream()>>>(
          dev_ins_data, dev_ins_col_data, static_cast<int>(inputs_col.size()),
          out_row, out_col, output->data<T>());
    }
  }
};

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */
template <typename T>
class ConcatGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const int axis,
                  std::vector<framework::Tensor>& outputs) {
    // TODO(zcd): Add input data validity checking
    int o_num = outputs.size();
    int out_row = 1;
    auto dim_0 = outputs[0].dims();
    for (int i = 0; i < axis; ++i) {
      out_row *= dim_0[i];
    }

    int out_col = outputs[0].numel() / out_row;
    int in_col = 0, in_row = out_row;
    bool sameShape = true;

    framework::Vector<int16_t> outputs_data(o_num * sizeof(T*) / 2);
    framework::Vector<int> outputs_cols(o_num + 1);
    T** outputs_ptr = reinterpret_cast<T**>(outputs_data.data());

    outputs_cols[0] = 0;
    for (int i = 0; i < o_num; ++i) {
      int t_col = outputs[i].numel() / out_row;
      if (sameShape) {
        if (t_col != out_col) sameShape = false;
      }
      in_col += t_col;
      outputs_cols[i + 1] = in_col;
      outputs_ptr[i] = outputs[i].data<T>();
    }

    T** dev_out_gpu_data =
        reinterpret_cast<T**>(outputs_data.CUDAMutableData(context.GetPlace()));

    // computation
    const int kThreadsPerBlock = 1024;
    int block_cols = kThreadsPerBlock;
    if (in_col < kThreadsPerBlock) {  // block_cols is aligned by 32.
      block_cols = ((in_col + 31) >> 5) << 5;
    }
    int block_rows = kThreadsPerBlock / block_cols;
    dim3 block_size = dim3(block_cols, block_rows, 1);

    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    int grid_cols =
        std::min((in_col + block_cols - 1) / block_cols, max_blocks);
    int grid_rows =
        std::min(max_blocks / grid_cols, std::max(out_row / block_rows, 1));
    dim3 grid_size = dim3(grid_cols, grid_rows, 1);

    if (sameShape) {
      KernelConcatGrad<<<grid_size, block_size, 0, context.stream()>>>(
          input.data<T>(), in_row, in_col, out_col, dev_out_gpu_data);
    } else {
      const int* dev_outs_col_data = outputs_cols.CUDAData(context.GetPlace());
      KernelConcatGrad<<<grid_size, block_size, 0, context.stream()>>>(
          input.data<T>(), in_row, in_col, dev_outs_col_data,
          static_cast<int>(outputs_cols.size()), dev_out_gpu_data);
    }
  }
};

template class ConcatFunctor<platform::CUDADeviceContext, int>;
template class ConcatFunctor<platform::CUDADeviceContext, int64_t>;
template class ConcatFunctor<platform::CUDADeviceContext, float>;
template class ConcatFunctor<platform::CUDADeviceContext, double>;

template class ConcatGradFunctor<platform::CUDADeviceContext, int>;
template class ConcatGradFunctor<platform::CUDADeviceContext, int64_t>;
template class ConcatGradFunctor<platform::CUDADeviceContext, float>;
template class ConcatGradFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
