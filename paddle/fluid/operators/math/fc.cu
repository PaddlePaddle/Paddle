#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/fc.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
struct FcTypeTraits;

template <>
struct FcTypeTraits<float> {
  typedef float4 Type;
};

template <>
struct FcTypeTraits<double> {
  typedef double4 Type;
};

template <typename T, bool DoRelu>
__global__ void bias_relu_v4(const int num, const T* bias, T* data, int K) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int bias_idx = tid % K;
    const T bias_ptr = bias[bias_idx];
    const T in_ptr = data[tid];
    T packed_val;
    packed_val.x = in_ptr.x + bias_ptr.x;
    packed_val.y = in_ptr.y + bias_ptr.y;
    packed_val.z = in_ptr.z + bias_ptr.z;
    packed_val.w = in_ptr.w + bias_ptr.w;
    if (DoRelu) {
      packed_val.x = fmaxf(0.f, packed_val.x);
      packed_val.y = fmaxf(0.f, packed_val.y);
      packed_val.z = fmaxf(0.f, packed_val.z);
      packed_val.w = fmaxf(0.f, packed_val.w);
    }
    data[tid] = packed_val;
  }
}

template <typename T, bool DoRelu, int BlockDim>
__global__ void InplaceAddReluKernel(const int N, const T* bias, T* data) {
  int offset = blockIdx.x * N;

  for (int i = threadIdx.x; i < N; i += BlockDim) {
    T temp;
#if __CUDA_ARCH__ >= 350
    temp = __ldg(data + offset + i) + __ldg(bias + i);
#else
    temp = data[offset + i] + bias[i];
#endif
    if (DoRelu) {
      data[offset + i] = static_cast<int>(temp > 0) * temp;
    } else {
      data[offset + i] = temp;
    }
  }
}

template <typename T>
class FCFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context, const int M,
                  const int N, const int K, const T* X, const T* W, T* Y,
                  const T* B = nullptr, bool relu = false,
                  bool padding_weights = false) {
    PADDLE_ENFORCE_EQ(
        padding_weights, false,
        platform::errors::PermissionDenied(
            "Weight padding in fc can not be used in GPU scope."));
    auto blas = math::GetBlas<platform::CUDADeviceContext, T>(context);
    blas.GEMM(false, false, M, N, K, static_cast<T>(1.0), X, K, W, N,
              static_cast<T>(0.0), Y, N);
    if (B == NULL) {
      return;
    }

    // M * N
    if (N % 4 == 0) {
      const int threads = 256;
      const int num = M * N / 4;
      const int blocks = (num + threads - 1) / threads;
      typedef typename FcTypeTraits<T>::Type trans_type;
      auto* bias_ptr_v4 = reinterpret_cast<const trans_type*>(B);
      auto* data_ptr_v4 = reinterpret_cast<trans_type*>(Y);
      if (relu) {
        bias_relu_v4<trans_type,
                     true><<<blocks, threads, 0, context.stream()>>>(
            num, bias_ptr_v4, data_ptr_v4, N / 4);
      } else {
        bias_relu_v4<trans_type,
                     false><<<blocks, threads, 0, context.stream()>>>(
            num, bias_ptr_v4, data_ptr_v4, N / 4);
      }
    } else {
      const int threads = 256;
      const int blocks = M;
      if (relu) {
        InplaceAddReluKernel<T, true,
                             threads><<<blocks, threads, 0, context.stream()>>>(
            N, B, Y);
      } else {
        InplaceAddReluKernel<T, false,
                             threads><<<blocks, threads, 0, context.stream()>>>(
            N, B, Y);
      }
    }
  }
};

template class FCFunctor<platform::CUDADeviceContext, float>;
template class FCFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
