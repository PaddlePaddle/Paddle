#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>

#include "paddle/fluid/operators/math/pooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(const int nthreads, const T* input_data,
                             const int channels, const int input_height,
                             const int input_width, const int output_height,
                             const int output_width, const int ksize_height,
                             const int ksize_width, const int stride_height,
                             const int stride_width, const int padding_height,
                             const int padding_width, PoolProcess pool_process,
                             bool exclusive, bool adaptive, T* output_data,
                             bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, c, batch_idx;
    if (!channel_last) { /*NCHW*/
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      c = (index / output_width / output_height) % channels;
      batch_idx = index / output_width / output_height / channels;
    } else { /*NHWC*/
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      batch_idx = index / channels / output_width / output_height;
    }

    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      hstart = ph * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = pw * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    if (!channel_last) {
      input_data += (batch_idx * channels + c) * input_height * input_width;
    } else {
      input_data += batch_idx * input_height * input_width * channels;
    }
    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        auto input_idx = channel_last ? (h * input_width + w) * channels + c
                                      : h * input_width + w;
        pool_process.compute(input_data[input_idx], &ele);
      }
    }
    int pool_size = (exclusive || adaptive) ? (hend - hstart) * (wend - wstart)
                                            : ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}
template <typename PoolProcess, typename T>
__global__ void KernelPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, bool adaptive, T* input_grad,
    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset, h_offset, offsetC, batch_idx;
    if (!channel_last) { /* NCHW */
      w_offset = index % input_width + padding_width;
      h_offset = (index / input_width) % input_height + padding_height;
      offsetC = (index / input_width / input_height) % channels;
      batch_idx = index / input_width / input_height / channels;
    } else { /* NHWC */
      offsetC = index % channels;
      w_offset = (index / channels) % input_width + padding_width;
      h_offset =
          (index / channels / input_width) % input_height + padding_height;
      batch_idx = index / channels / input_width / input_height;
    }

    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      phstart = AdaptStartIndex(h_offset, output_height, input_height);
      phend = AdaptEndIndex(h_offset, output_height, input_height);

      pwstart = AdaptStartIndex(w_offset, output_width, input_width);
      pwend = AdaptEndIndex(w_offset, output_width, input_width);
    } else {
      phstart = (h_offset < ksize_height)
                    ? 0
                    : (h_offset - ksize_height) / stride_height + 1;
      pwstart = (w_offset < ksize_width)
                    ? 0
                    : (w_offset - ksize_width) / stride_width + 1;
      phend = min(h_offset / stride_height + 1, output_height);
      pwend = min(w_offset / stride_width + 1, output_width);
    }
    T gradient = static_cast<T>(0.0);
    T input = input_data[index];

    int output_stride;
    if (!channel_last) {
      output_stride =
          (batch_idx * channels + offsetC) * output_height * output_width;
    } else {
      output_stride = batch_idx * output_height * output_width * channels;
    }

    output_data += output_stride;
    output_grad += output_stride;

    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int pool_size;
        if (adaptive) {
          pool_size = static_cast<int>(ceil(static_cast<double>(input_height) /
                                            ksize_height)) *
                      static_cast<int>(
                          ceil(static_cast<double>(input_width) / ksize_width));
        } else {
          int hstart = ph * stride_height - padding_height;
          int wstart = pw * stride_width - padding_width;
          int hend = min(hstart + ksize_height, input_height);
          int wend = min(wstart + ksize_width, input_width);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          pool_size = exclusive ? (hend - hstart) * (wend - wstart)
                                : ksize_height * ksize_width;
        }

        int output_sub_idx = channel_last
                                 ? (ph * output_width + pw) * channels + offsetC
                                 : ph * output_width + pw;
        pool_process.compute(input, output_data[output_sub_idx],
                             output_grad[output_sub_idx],
                             static_cast<T>(1.0 / pool_size), &gradient);
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    T* input_grad, bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, c, batch_idx;
    if (!channel_last) { /* NCHW */
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      c = (index / output_width / output_height) % channels;
      batch_idx = index / output_width / output_height / channels;
    } else { /* NHWC */
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      batch_idx = index / channels / output_width / output_height;
    }
    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    int input_stride;
    if (!channel_last) {
      input_stride = (batch_idx * channels + c) * input_height * input_width;
    } else {
      input_stride = batch_idx * input_height * input_width * channels;
    }
    input_data += input_stride;
    input_grad += input_stride;

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        int input_data_idx = channel_last ? (h * input_width + w) * channels + c
                                          : h * input_width + w;
        if (ele == input_data[input_data_idx]) {
          maxIndex = input_data_idx;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIndex, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
void Pool2dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input, const std::vector<int>& input_shape,
    const std::vector<int>& output_shape, const std::vector<int>& ksize,
    const std::vector<int>& strides, const std::vector<int>& paddings,
    bool exclusive, bool adaptive, T* output, gpuStream_t stream,
    PoolProcess pool_compute) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_height = input_shape[2];
  const int input_width = input_shape[3];
  const int output_channels = output_shape[1];
  const int output_height = output_shape[2];
  const int output_width = output_shape[3];
  const int ksize_height = ksize[0];
  const int ksize_width = ksize[1];
  const int stride_height = strides[0];
  const int stride_width = strides[1];
  const int padding_height = paddings[0];
  const int padding_width = paddings[1];

  int nthreads = batch_size * output_channels * output_height * output_width;
  int blocks = (nthreads + 1024 - 1) / 1024;
  dim3 threads(1024, 1);
  dim3 grid(blocks, 1);

  KernelPool2D<PoolProcess, T><<<grid, threads, 0, stream>>>(
      nthreads, input, input_channels, input_height, input_width, output_height,
      output_width, ksize_height, ksize_width, stride_height, stride_width,
      padding_height, padding_width, pool_compute, exclusive, adaptive, output);
}

/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* output,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* output, PoolProcess pool_process) {
    bool channel_last = (data_format == "NHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output->dims()[3] : output->dims()[1];
    const int output_height =
        channel_last ? output->dims()[1] : output->dims()[2];
    const int output_width =
        channel_last ? output->dims()[2] : output->dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data, channel_last);
  }
};
/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* input_grad,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        pool_process, exclusive, adaptive, input_grad_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* input_grad, PoolProcess pool_process) {
    bool channel_last = (data_format == "NHWC");

    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output.dims()[3] : output.dims()[1];
    const int output_height =
        channel_last ? output.dims()[1] : output.dims()[2];
    const int output_width = channel_last ? output.dims()[2] : output.dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();

    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        pool_process, exclusive, adaptive, input_grad_data, channel_last);
  }
};

/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename T>
class MaxPool2dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        input_grad_data);
  }
  void operator()(
      const platform::CUDADeviceContext& context,
      const framework::Tensor& input, const framework::Tensor& output,
      const framework::Tensor& output_grad, const std::vector<int>& ksize,
      const std::vector<int>& strides, const std::vector<int>& paddings,
      const std::string data_format, framework::Tensor* input_grad) {
    bool channel_last = (data_format == "NHWC");

    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output.dims()[3] : output.dims()[1];
    const int output_height =
        channel_last ? output.dims()[1] : output.dims()[2];
    const int output_width = channel_last ? output.dims()[2] : output.dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        input_grad_data, channel_last);
  }
};

template class Pool2dDirectCUDAFunctor<paddle::operators::math::MaxPool<float>,
                                       float>;
template class Pool2dDirectCUDAFunctor<paddle::operators::math::AvgPool<float>,
                                       float>;

template class MaxPool2dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool2dGradFunctor<platform::CUDADeviceContext, double>;
template class MaxPool2dGradFunctor<platform::CUDADeviceContext,
                                    paddle::platform::float16>;

template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template class Pool2dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool2dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool2dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool2dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3D(
    const int nthreads, const T* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, bool adaptive, T* output_data,
    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, pd, c, batch_idx;
    if (!channel_last) {
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      pd = (index / output_width / output_height) % output_depth;
      c = (index / output_width / output_height / output_depth) % channels;
      batch_idx =
          index / output_width / output_height / output_depth / channels;
    } else {
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      pd = (index / channels / output_width / output_height) % output_depth;
      batch_idx =
          index / channels / output_width / output_height / output_depth;
    }

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = AdaptStartIndex(pd, input_depth, output_depth);
      dend = AdaptEndIndex(pd, input_depth, output_depth);

      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    int input_data_stride;
    if (!channel_last) { /* NCDHW */
      input_data_stride =
          (batch_idx * channels + c) * input_depth * input_height * input_width;
    } else { /* NDHWC */
      input_data_stride =
          batch_idx * input_depth * input_height * input_width * channels;
    }
    input_data += input_data_stride;

    T ele = pool_process.initial();
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          auto input_data_idx =
              channel_last
                  ? ((d * input_height + h) * input_width + w) * channels + c
                  : (d * input_height + h) * input_width + w;
          pool_process.compute(input_data[input_data_idx], &ele);
        }
      }
    }
    int pool_size = (exclusive || adaptive)
                        ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                        : ksize_depth * ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, PoolProcess pool_process,
    bool exclusive, bool adaptive, T* input_grad, bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset, h_offset, d_offset, offsetC, batch_idx;
    if (!channel_last) { /* "NCDHW" */
      w_offset = index % input_width + padding_width;
      h_offset = (index / input_width) % input_height + padding_height;
      d_offset =
          (index / input_width / input_height) % input_depth + padding_depth;
      offsetC = (index / input_width / input_height / input_depth) % channels;
      batch_idx = index / input_width / input_height / input_depth / channels;

    } else { /* "NDHWC" */
      offsetC = index % channels;
      w_offset = (index / channels) % input_width + padding_width;
      h_offset =
          (index / channels / input_width) % input_height + padding_height;
      d_offset = (index / channels / input_width / input_height) % input_depth +
                 padding_depth;
      batch_idx = index / channels / input_width / input_height / input_depth;
    }

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = AdaptStartIndex(d_offset, output_depth, input_depth);
      pdend = AdaptEndIndex(d_offset, output_depth, input_depth);

      phstart = AdaptStartIndex(h_offset, output_height, input_height);
      phend = AdaptEndIndex(h_offset, output_height, input_height);

      pwstart = AdaptStartIndex(w_offset, output_width, input_width);
      pwend = AdaptEndIndex(w_offset, output_width, input_width);
    } else {
      pdstart = (d_offset < ksize_depth)
                    ? 0
                    : (d_offset - ksize_depth) / stride_depth + 1;
      phstart = (h_offset < ksize_height)
                    ? 0
                    : (h_offset - ksize_height) / stride_height + 1;
      pwstart = (w_offset < ksize_width)
                    ? 0
                    : (w_offset - ksize_width) / stride_width + 1;
      pdend = min((d_offset) / stride_depth + 1, output_depth);
      phend = min((h_offset) / stride_height + 1, output_height);
      pwend = min((w_offset) / stride_width + 1, output_width);
    }

    T gradient = static_cast<T>(0.0);
    T input = input_data[index];

    int output_stride;
    if (!channel_last) {
      output_stride = (batch_idx * channels + offsetC) * output_depth *
                      output_height * output_width;
    } else {
      output_stride =
          batch_idx * output_depth * output_height * output_width * channels;
    }
    output_data += output_stride;
    output_grad += output_stride;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int pool_size;
          if (adaptive) {
            pool_size =
                static_cast<int>(
                    ceil(static_cast<double>(input_depth) / ksize_depth)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_height) / ksize_height)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_width) / ksize_width));
          } else {
            int dstart = pd * stride_depth - padding_depth;
            int hstart = ph * stride_height - padding_height;
            int wstart = pw * stride_width - padding_width;
            int dend = min(dstart + ksize_depth, input_depth);
            int hend = min(hstart + ksize_height, input_height);
            int wend = min(wstart + ksize_width, input_width);
            dstart = max(dstart, 0);
            hstart = max(hstart, 0);
            wstart = max(wstart, 0);
            pool_size =
                exclusive ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                          : ksize_depth * ksize_height * ksize_width;
          }

          int output_sub_idx =
              channel_last
                  ? ((pd * output_height + ph) * output_width + pw) * channels +
                        offsetC
                  : (pd * output_height + ph) * output_width + pw;

          pool_process.compute(input, output_data[output_sub_idx],
                               output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size), &gradient);
        }
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, T* input_grad,
    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, pd, c, batch_idx;

    if (!channel_last) { /*NCDHW*/
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      pd = (index / output_width / output_height) % output_depth;
      c = (index / output_width / output_height / output_depth) % channels;
      batch_idx =
          index / output_width / output_height / output_depth / channels;
    } else { /*NDHWC*/
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      pd = (index / channels / output_width / output_height) % output_depth;
      batch_idx =
          index / channels / output_width / output_height / output_depth;
    }

    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;

    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);

    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);

    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;

    int input_stride;
    if (!channel_last) {
      input_stride =
          (batch_idx * channels + c) * input_depth * input_height * input_width;
    } else {
      input_stride =
          batch_idx * input_depth * input_height * input_width * channels;
    }
    input_data += input_stride;
    input_grad += input_stride;
    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          int input_data_idx =
              channel_last
                  ? ((d * input_height + h) * input_width + w) * channels + c
                  : (d * input_height + h) * input_width + w;
          if (ele == input_data[input_data_idx]) {
            stop = true;
            maxIdx = input_data_idx;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

/*
 * Tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* output,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* output, PoolProcess pool_process) {
    bool channel_last = (data_format == "NDHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output->dims()[4] : output->dims()[1];
    const int output_depth =
        channel_last ? output->dims()[1] : output->dims()[2];
    const int output_height =
        channel_last ? output->dims()[2] : output->dims()[3];
    const int output_width =
        channel_last ? output->dims()[3] : output->dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data, channel_last);
  }
};

/*
 * Tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* input_grad,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, pool_process, exclusive, adaptive, input_grad_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* input_grad, PoolProcess pool_process) {
    bool channel_last = (data_format == "NDHWC");

    const int batch_size = input.dims()[0];
    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output.dims()[4] : output.dims()[1];
    const int output_depth = channel_last ? output.dims()[1] : output.dims()[2];
    const int output_height =
        channel_last ? output.dims()[2] : output.dims()[3];
    const int output_width = channel_last ? output.dims()[3] : output.dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, pool_process, exclusive, adaptive, input_grad_data,
        channel_last);  // add channel_last
  }
};

/*
 * tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <class T>
class MaxPool3dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, input_grad_data);
  }
  void operator()(
      const platform::CUDADeviceContext& context,
      const framework::Tensor& input, const framework::Tensor& output,
      const framework::Tensor& output_grad, const std::vector<int>& ksize,
      const std::vector<int>& strides, const std::vector<int>& paddings,
      const std::string data_format, framework::Tensor* input_grad) {
    bool channel_last = (data_format == "NDHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output.dims()[4] : output.dims()[1];
    const int output_depth = channel_last ? output.dims()[1] : output.dims()[2];
    const int output_height =
        channel_last ? output.dims()[2] : output.dims()[3];
    const int output_width = channel_last ? output.dims()[3] : output.dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, input_grad_data, channel_last);  // add channel_last
  }
};

template class MaxPool3dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool3dGradFunctor<platform::CUDADeviceContext, double>;
template class MaxPool3dGradFunctor<platform::CUDADeviceContext,
                                    paddle::platform::float16>;

template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template class Pool3dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool3dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool3dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool3dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;

template <typename T1, typename T2>
__global__ void KernelMaxPool2dWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, bool adaptive, T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      hstart = ph * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = pw * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T1 ele = -FLT_MAX;
    int max_index = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_index = h * input_width + w;
        if (ele < input_data[input_index]) {
          max_index = input_index;
          ele = input_data[input_index];
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool2DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask_data,
    const int channels, const int input_height, const int input_width,
    const int output_height, const int output_width, const int ksize_height,
    const int ksize_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, bool adaptive,
    T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset = index % input_width;
    int h_offset = (index / input_width) % input_height;
    int offsetC = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      phstart = h_offset * output_height / input_height;
      phend =
          min((h_offset + 1) * output_height / input_height + 1, output_height);
      pwstart = w_offset * output_width / input_width;
      pwend =
          min((w_offset + 1) * output_width / input_width + 1, output_width);
    } else {
      phstart =
          (h_offset + padding_height < ksize_height)
              ? 0
              : (h_offset + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (w_offset + padding_width < ksize_width)
              ? 0
              : (w_offset + padding_width - ksize_width) / stride_width + 1;
      phend =
          min((h_offset + padding_height) / stride_height + 1, output_height);
      pwend = min((w_offset + padding_width) / stride_width + 1, output_width);
    }

    T1 gradient = 0;
    int input_current_featuremap_idx = h_offset * input_width + w_offset;
    int output_idx =
        (batch_idx * channels + offsetC) * output_height * output_width;

    mask_data += output_idx;
    output_grad += output_idx;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_data[ph * output_width + pw] == input_current_featuremap_idx)
          gradient += output_grad[ph * output_width + pw];
      }
    }
    input_grad[index] = gradient;
  }
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* output, framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2dWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, adaptive, output_data,
        mask_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_height = input_grad->dims()[2];
    const int input_width = input_grad->dims()[3];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T2* mask_data = mask.data<T2>();
    const T1* output_grad_data = output_grad.data<T1>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_height,
        input_width, output_height, output_width, ksize_height, ksize_width,
        stride_height, stride_width, padding_height, padding_width, adaptive,
        input_grad_data);
  }
};

template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    bool adaptive, T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = AdaptStartIndex(pd, input_depth, output_depth);
      dend = AdaptEndIndex(pd, input_depth, output_depth);

      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    T1 ele = -FLT_MAX;
    int max_index = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (ele < input_data[(d * input_height + h) * input_width + w]) {
            max_index = (d * input_height + h) * input_width + w;
            ele = input_data[max_index];
          }
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask,
    const int channels, const int input_depth, const int input_height,
    const int input_width, const int output_depth, const int output_height,
    const int output_width, const int ksize_depth, const int ksize_height,
    const int ksize_width, const int stride_depth, const int stride_height,
    const int stride_width, const int padding_depth, const int padding_height,
    const int padding_width, bool adaptive, T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset = index % input_width;
    int h_offset = (index / input_width) % input_height;
    int d_offset = (index / input_width / input_height) % input_depth;
    int offsetC = (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = d_offset * output_depth / input_depth;
      pdend =
          min((d_offset + 1) * output_depth / input_depth + 1, output_depth);
      phstart = h_offset * output_height / input_height;
      phend =
          min((h_offset + 1) * output_height / input_height + 1, output_height);
      pwstart = w_offset * output_width / input_width;
      pwend =
          min((w_offset + 1) * output_width / input_width + 1, output_width);
    } else {
      pdstart =
          (d_offset + padding_depth < ksize_depth)
              ? 0
              : (d_offset + padding_depth - ksize_depth) / stride_depth + 1;
      phstart =
          (h_offset + padding_height < ksize_height)
              ? 0
              : (h_offset + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (w_offset + padding_width < ksize_width)
              ? 0
              : (w_offset + padding_width - ksize_width) / stride_width + 1;
      pdend = min((d_offset + padding_depth) / stride_depth + 1, output_depth);
      phend =
          min((h_offset + padding_height) / stride_height + 1, output_height);
      pwend = min((w_offset + padding_width) / stride_width + 1, output_width);
    }

    T1 gradient = 0;
    int input_current_feature_map_idx =
        (d_offset * input_height + h_offset) * input_width + w_offset;
    int output_idx = (batch_idx * channels + offsetC) * output_depth *
                     output_height * output_width;
    mask += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[(pd * output_height + ph) * output_width + pw] ==
              input_current_feature_map_idx)
            gradient +=
                output_grad[(pd * output_height + ph) * output_width + pw];
        }
      }
    }
    input_grad[index] = gradient;
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* output, framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, adaptive, output_data,
        mask_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_depth = input_grad->dims()[2];
    const int input_height = input_grad->dims()[3];
    const int input_width = input_grad->dims()[4];
    const int output_depth = output_grad.dims()[2];
    const int output_height = output_grad.dims()[3];
    const int output_width = output_grad.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* output_grad_data = output_grad.data<T1>();
    const T2* mask_data = mask.data<T2>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_depth,
        input_height, input_width, output_depth, output_height, output_width,
        ksize_depth, ksize_height, ksize_width, stride_depth, stride_height,
        stride_width, padding_depth, padding_height, padding_width, adaptive,
        input_grad_data);
  }
};

template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
