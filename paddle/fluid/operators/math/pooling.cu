#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "paddle/fluid/operators/math/pooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

__device__ __forceinline__ int ADAPT_START_INDEX(int ph, int input_size,
                                                 int output_size) {
  return static_cast<int>(
      floor(static_cast<double>(ph * input_size) / output_size));
}

__device__ __forceinline__ int ADAPT_END_INDEX(int ph, int input_size,
                                               int output_size) {
  return static_cast<int>(
      ceil(static_cast<double>((ph + 1) * input_size) / output_size));
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(const int nthreads, const T* input_data,
                             const int channels, const int input_height,
                             const int input_width, const int output_height,
                             const int output_width, const int ksize_height,
                             const int ksize_width, const int stride_height,
                             const int stride_width, const int padding_height,
                             const int padding_width, PoolProcess pool_process,
                             bool exclusive, bool adaptive, T* output_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      hstart = ADAPT_START_INDEX(ph, input_height, output_height);
      hend = ADAPT_END_INDEX(ph, input_height, output_height);

      wstart = ADAPT_START_INDEX(pw, input_width, output_width);
      wend = ADAPT_END_INDEX(pw, input_width, output_width);
    } else {
      hstart = ph * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = pw * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        pool_process.compute(input_data[h * input_width + w], &ele);
      }
    }
    int pool_size = (exclusive || adaptive) ? (hend - hstart) * (wend - wstart)
                                            : ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, bool adaptive, T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetC = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      phstart = offsetH * output_height / input_height;
      phend =
          min((offsetH + 1) * output_height / input_height + 1, output_height);
      pwstart = offsetW * output_width / input_width;
      pwend = min((offsetW + 1) * output_width / input_width + 1, output_width);
    } else {
      phstart = (offsetH < ksize_height)
                    ? 0
                    : (offsetH - ksize_height) / stride_height + 1;
      pwstart = (offsetW < ksize_width)
                    ? 0
                    : (offsetW - ksize_width) / stride_width + 1;
      phend = min(offsetH / stride_height + 1, output_height);
      pwend = min(offsetW / stride_width + 1, output_width);
    }
    T gradient = 0;
    T input = input_data[index];
    int output_idx =
        (batch_idx * channels + offsetC) * output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int pool_size;
        if (adaptive) {
          pool_size = static_cast<int>(ceil(static_cast<double>(input_height) /
                                            ksize_height)) *
                      static_cast<int>(
                          ceil(static_cast<double>(input_width) / ksize_width));
        } else {
          int hstart = ph * stride_height - padding_height;
          int wstart = pw * stride_width - padding_width;
          int hend = min(hstart + ksize_height, input_height);
          int wend = min(wstart + ksize_width, input_width);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          pool_size = exclusive ? (hend - hstart) * (wend - wstart)
                                : ksize_height * ksize_width;
        }
        int output_sub_idx = ph * output_width + pw;
        pool_process.compute(input, output_data[output_sub_idx],
                             output_grad[output_sub_idx],
                             static_cast<T>(1.0 / pool_size), &gradient);
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    input_grad += (batch_idx * channels + c) * input_height * input_width;

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        if (ele == input_data[h * input_width + w]) {
          maxIndex = h * input_width + w;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIndex, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
void Pool2dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input, const std::vector<int>& input_shape,
    const std::vector<int>& output_shape, const std::vector<int>& ksize,
    const std::vector<int>& strides, const std::vector<int>& paddings,
    PoolProcess pool_compute, bool exclusive, T* output, hipStream_t stream) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_height = input_shape[2];
  const int input_width = input_shape[3];
  const int output_channels = output_shape[1];
  const int output_height = output_shape[2];
  const int output_width = output_shape[3];
  const int ksize_height = ksize[0];
  const int ksize_width = ksize[1];
  const int stride_height = strides[0];
  const int stride_width = strides[1];
  const int padding_height = paddings[0];
  const int padding_width = paddings[1];

  int nthreads = batch_size * output_channels * output_height * output_width;
  int blocks = (nthreads + 1024 - 1) / 1024;
  dim3 threads(1024, 1);
  dim3 grid(blocks, 1);

  KernelPool2D<PoolProcess, T><<<grid, threads, 0, stream>>>(
      nthreads, input, input_channels, input_height, input_width, output_height,
      output_width, ksize_height, ksize_width, stride_height, stride_width,
      padding_height, padding_width, pool_compute, exclusive, false, output);
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, bool adaptive, framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        pool_process, exclusive, adaptive, input_grad_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T>
class MaxPool2dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        input_grad_data);
  }
};

template class Pool2dDirectCUDAFunctor<paddle::operators::math::MaxPool<float>,
                                       float>;
template class Pool2dDirectCUDAFunctor<paddle::operators::math::AvgPool<float>,
                                       float>;

template class MaxPool2dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool2dGradFunctor<platform::CUDADeviceContext, double>;

template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3D(
    const int nthreads, const T* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, bool adaptive, T* output_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = ADAPT_START_INDEX(pd, input_depth, output_depth);
      dend = ADAPT_END_INDEX(pd, input_depth, output_depth);

      hstart = ADAPT_START_INDEX(ph, input_height, output_height);
      hend = ADAPT_END_INDEX(ph, input_height, output_height);

      wstart = ADAPT_START_INDEX(pw, input_width, output_width);
      wend = ADAPT_END_INDEX(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }
    T ele = pool_process.initial();
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          pool_process.compute(
              input_data[(d * input_height + h) * input_width + w], &ele);
        }
      }
    }
    int pool_size = (exclusive || adaptive)
                        ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                        : ksize_depth * ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, PoolProcess pool_process,
    bool exclusive, bool adaptive, T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetD =
        (index / input_width / input_height) % input_depth + padding_depth;
    int offsetC = (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = offsetD * output_depth / input_depth;
      pdend = min((offsetD + 1) * output_depth / input_depth + 1, output_depth);
      phstart = offsetH * output_height / input_height;
      phend =
          min((offsetH + 1) * output_height / input_height + 1, output_height);
      pwstart = offsetW * output_width / input_width;
      pwend = min((offsetW + 1) * output_width / input_width + 1, output_width);
    } else {
      pdstart = (offsetD < ksize_depth)
                    ? 0
                    : (offsetD - ksize_depth) / stride_depth + 1;
      phstart = (offsetH < ksize_height)
                    ? 0
                    : (offsetH - ksize_height) / stride_height + 1;
      pwstart = (offsetW < ksize_width)
                    ? 0
                    : (offsetW - ksize_width) / stride_width + 1;
      pdend = min((offsetD) / stride_depth + 1, output_depth);
      phend = min((offsetH) / stride_height + 1, output_height);
      pwend = min((offsetW) / stride_width + 1, output_width);
    }

    T gradient = 0;
    T input = input_data[index];
    int output_idx = (batch_idx * channels + offsetC) * output_depth *
                     output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int pool_size;
          if (adaptive) {
            pool_size =
                static_cast<int>(
                    ceil(static_cast<double>(input_depth) / ksize_depth)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_height) / ksize_height)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_width) / ksize_width));
          } else {
            int dstart = pd * stride_depth - padding_depth;
            int hstart = ph * stride_height - padding_height;
            int wstart = pw * stride_width - padding_width;
            int dend = min(dstart + ksize_depth, input_depth);
            int hend = min(hstart + ksize_height, input_height);
            int wend = min(wstart + ksize_width, input_width);
            dstart = max(dstart, 0);
            hstart = max(hstart, 0);
            wstart = max(wstart, 0);
            pool_size =
                exclusive ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                          : ksize_depth * ksize_height * ksize_width;
          }
          int output_sub_idx = (pd * output_height + ph) * output_width + pw;
          pool_process.compute(input, output_data[output_sub_idx],
                               output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size), &gradient);
        }
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    input_grad +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          if (ele == input_data[(d * input_height + h) * input_width + w]) {
            stop = true;
            maxIdx = (d * input_height + h) * input_width + w;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, bool adaptive, framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, pool_process, exclusive, adaptive, input_grad_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <class T>
class MaxPool3dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, input_grad_data);
  }
};

template class MaxPool3dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool3dGradFunctor<platform::CUDADeviceContext, double>;

template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template <typename T1, typename T2>
__global__ void KernelMaxPool2dWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, bool adaptive, T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      hstart = ADAPT_START_INDEX(ph, input_height, output_height);
      hend = ADAPT_END_INDEX(ph, input_height, output_height);

      wstart = ADAPT_START_INDEX(pw, input_width, output_width);
      wend = ADAPT_END_INDEX(pw, input_width, output_width);
    } else {
      hstart = ph * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = pw * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T1 ele = -FLT_MAX;
    int max_index = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_index = h * input_width + w;
        if (ele < input_data[input_index]) {
          max_index = input_index;
          ele = input_data[input_index];
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool2DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask_data,
    const int channels, const int input_height, const int input_width,
    const int output_height, const int output_width, const int ksize_height,
    const int ksize_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, bool adaptive,
    T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width;
    int offsetH = (index / input_width) % input_height;
    int offsetC = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      phstart = offsetH * output_height / input_height;
      phend =
          min((offsetH + 1) * output_height / input_height + 1, output_height);
      pwstart = offsetW * output_width / input_width;
      pwend = min((offsetW + 1) * output_width / input_width + 1, output_width);
    } else {
      phstart =
          (offsetH + padding_height < ksize_height)
              ? 0
              : (offsetH + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (offsetW + padding_width < ksize_width)
              ? 0
              : (offsetW + padding_width - ksize_width) / stride_width + 1;
      phend =
          min((offsetH + padding_height) / stride_height + 1, output_height);
      pwend = min((offsetW + padding_width) / stride_width + 1, output_width);
    }

    T1 gradient = 0;
    int input_current_featuremap_idx = offsetH * input_width + offsetW;
    int output_idx =
        (batch_idx * channels + offsetC) * output_height * output_width;

    mask_data += output_idx;
    output_grad += output_idx;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_data[ph * output_width + pw] == input_current_featuremap_idx)
          gradient += output_grad[ph * output_width + pw];
      }
    }
    input_grad[index] = gradient;
  }
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* output, framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2dWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, adaptive, output_data,
        mask_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_height = input_grad->dims()[2];
    const int input_width = input_grad->dims()[3];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T2* mask_data = mask.data<T2>();
    const T1* output_grad_data = output_grad.data<T1>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_height,
        input_width, output_height, output_width, ksize_height, ksize_width,
        stride_height, stride_width, padding_height, padding_width, adaptive,
        input_grad_data);
  }
};

template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    bool adaptive, T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = ADAPT_START_INDEX(pd, input_depth, output_depth);
      dend = ADAPT_END_INDEX(pd, input_depth, output_depth);

      hstart = ADAPT_START_INDEX(ph, input_height, output_height);
      hend = ADAPT_END_INDEX(ph, input_height, output_height);

      wstart = ADAPT_START_INDEX(pw, input_width, output_width);
      wend = ADAPT_END_INDEX(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    T1 ele = -FLT_MAX;
    int max_index = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (ele < input_data[(d * input_height + h) * input_width + w]) {
            max_index = (d * input_height + h) * input_width + w;
            ele = input_data[max_index];
          }
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask,
    const int channels, const int input_depth, const int input_height,
    const int input_width, const int output_depth, const int output_height,
    const int output_width, const int ksize_depth, const int ksize_height,
    const int ksize_width, const int stride_depth, const int stride_height,
    const int stride_width, const int padding_depth, const int padding_height,
    const int padding_width, bool adaptive, T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width;
    int offsetH = (index / input_width) % input_height;
    int offsetD = (index / input_width / input_height) % input_depth;
    int offsetC = (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = offsetD * output_depth / input_depth;
      pdend = min((offsetD + 1) * output_depth / input_depth + 1, output_depth);
      phstart = offsetH * output_height / input_height;
      phend =
          min((offsetH + 1) * output_height / input_height + 1, output_height);
      pwstart = offsetW * output_width / input_width;
      pwend = min((offsetW + 1) * output_width / input_width + 1, output_width);
    } else {
      pdstart =
          (offsetD + padding_depth < ksize_depth)
              ? 0
              : (offsetD + padding_depth - ksize_depth) / stride_depth + 1;
      phstart =
          (offsetH + padding_height < ksize_height)
              ? 0
              : (offsetH + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (offsetW + padding_width < ksize_width)
              ? 0
              : (offsetW + padding_width - ksize_width) / stride_width + 1;
      pdend = min((offsetD + padding_depth) / stride_depth + 1, output_depth);
      phend =
          min((offsetH + padding_height) / stride_height + 1, output_height);
      pwend = min((offsetW + padding_width) / stride_width + 1, output_width);
    }

    T1 gradient = 0;
    int input_current_feature_map_idx =
        (offsetD * input_height + offsetH) * input_width + offsetW;
    int output_idx = (batch_idx * channels + offsetC) * output_depth *
                     output_height * output_width;
    mask += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[(pd * output_height + ph) * output_width + pw] ==
              input_current_feature_map_idx)
            gradient +=
                output_grad[(pd * output_height + ph) * output_width + pw];
        }
      }
    }
    input_grad[index] = gradient;
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* output, framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, adaptive, output_data,
        mask_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_depth = input_grad->dims()[2];
    const int input_height = input_grad->dims()[3];
    const int input_width = input_grad->dims()[4];
    const int output_depth = output_grad.dims()[2];
    const int output_height = output_grad.dims()[3];
    const int output_width = output_grad.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* output_grad_data = output_grad.data<T1>();
    const T2* mask_data = mask.data<T2>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_depth,
        input_height, input_width, output_depth, output_height, output_width,
        ksize_depth, ksize_height, ksize_width, stride_depth, stride_height,
        stride_width, padding_depth, padding_height, padding_width, adaptive,
        input_grad_data);
  }
};

template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
