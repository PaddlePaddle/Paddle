#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>

#include "paddle/fluid/operators/math/pooling.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/fast_divmod.h"

namespace paddle {
namespace operators {
namespace math {

struct FastDivModForPooling {
 public:
  platform::FastDivMod channel;
  platform::FastDivMod width;
  platform::FastDivMod height;

  explicit HOSTDEVICE FastDivModForPooling(const int channels,
                                           const int output_width,
                                           const int output_height) {
    channel = platform::FastDivMod(channels);
    width = platform::FastDivMod(output_width);
    height = platform::FastDivMod(output_height);
  }
};

struct FastDivModForPoolingWithMoreStaff {
 public:
  platform::FastDivMod channel;
  platform::FastDivMod width;
  platform::FastDivMod height;
  platform::FastDivMod ksize_w;
  platform::FastDivMod ksize_h;
  platform::FastDivMod stride_w;
  platform::FastDivMod stride_h;

  explicit HOSTDEVICE FastDivModForPoolingWithMoreStaff(
      const int channels, const int input_width, const int input_height,
      const int ksize_width, const int ksize_height, const int stride_width,
      const int stride_height) {
    channel = platform::FastDivMod(channels);
    width = platform::FastDivMod(input_width);
    height = platform::FastDivMod(input_height);
    ksize_w = platform::FastDivMod(ksize_width);
    ksize_h = platform::FastDivMod(ksize_height);
    stride_w = platform::FastDivMod(stride_width);
    stride_h = platform::FastDivMod(stride_height);
  }
};

template <typename FastDivModForPooling>
__device__ void OffsetPreparationFor4Dimension(
    int index, bool channel_last, FastDivModForPooling divmods,
    const int pad_width, const int pad_height, const int aux_width,
    const int aux_height, int* w_offset, int* h_offset, int* c_offset,
    int* stride) {
  if (!channel_last) { /* NCHW */
    auto input_width_divmod = divmods.width.Divmod(index);
    auto input_height_divmod = divmods.height.Divmod(input_width_divmod.val[0]);
    auto channel_divmod = divmods.channel.Divmod(input_height_divmod.val[0]);
    *w_offset = input_width_divmod.val[1] + pad_width;
    *h_offset = input_height_divmod.val[1] + pad_height;
    *c_offset = channel_divmod.val[1];
    *stride = (channel_divmod.val[0] * divmods.channel.divisor + *c_offset) *
              aux_height * aux_width;
  } else { /* NHWC */
    auto c_divmod = divmods.channel.Divmod(index);
    auto input_width_divmod = divmods.width.Divmod(c_divmod.val[0]);
    auto input_height_divmod = divmods.height.Divmod(input_width_divmod.val[0]);
    *c_offset = c_divmod.val[1];
    *w_offset = input_width_divmod.val[1] + pad_width;
    *h_offset = input_height_divmod.val[1] + pad_height;
    *stride = input_height_divmod.val[0] * aux_height * aux_width *
              divmods.channel.divisor;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(
    const int nthreads, const T* input_data, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, FastDivModForPooling divmods,
    PoolProcess pool_process, bool exclusive, bool adaptive, T* output_data,
    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int hstart, hend, wstart, wend;
    int w_offset, h_offset, c_offset, input_offset;
    OffsetPreparationFor4Dimension<FastDivModForPooling>(
        index, channel_last, divmods, 0, 0, input_width, input_height,
        &w_offset, &h_offset, &c_offset, &input_offset);
    input_data += input_offset;

    if (adaptive) {
      hstart = AdaptStartIndex(h_offset, input_height, output_height);
      hend = AdaptEndIndex(h_offset, input_height, output_height);
      wstart = AdaptStartIndex(w_offset, input_width, output_width);
      wend = AdaptEndIndex(w_offset, input_width, output_width);
    } else {
      hstart = h_offset * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);
      wstart = w_offset * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        auto input_idx = channel_last
                             ? (h * input_width + w) * channels + c_offset
                             : h * input_width + w;
        pool_process.compute(input_data[input_idx], &ele);
      }
    }
    int pool_size = (exclusive || adaptive) ? (hend - hstart) * (wend - wstart)
                                            : ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename T, typename PoolProcess>
__global__ void KernelPool2DGrad(
    const int nthreads, const T* __restrict__ input_data,
    const T* __restrict__ output_data, const const T* __restrict__ output_grad,
    const int output_width, const int output_height, const int input_width,
    const int input_height, const int ksize_width, const int ksize_height,
    const int stride_width, const int stride_height, const int padding_width,
    const int padding_height, FastDivModForPoolingWithMoreStaff divmods,
    PoolProcess pool_process, bool exclusive, bool adaptive,
    T* __restrict__ input_grad, bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    T input = static_cast<T>(0);
    T input_grad_data = static_cast<T>(0);
    int phstart, phend, pwstart, pwend;
    int w_offset, h_offset, c_offset, output_offset;
    OffsetPreparationFor4Dimension<>(index, channel_last, divmods,
                                     padding_width, padding_height,
                                     output_width, output_height, &w_offset,
                                     &h_offset, &c_offset, &output_offset);
    if (pool_process.use_x) {
      input = input_data[index];
      output_data += output_offset;
    }
    output_grad += output_offset;

    if (adaptive) {
      auto tmp_phend = divmods.height.Divmod((h_offset + 1) * output_height);
      auto tmp_pwend = divmods.width.Divmod((w_offset + 1) * output_width);
      phstart = divmods.height.Div(h_offset * output_height);
      pwstart = divmods.width.Div(w_offset * output_width);
      phend = tmp_phend.val[1] > 0 ? tmp_phend.val[0] + 1 : tmp_phend.val[0];
      pwend = tmp_pwend.val[1] > 0 ? tmp_pwend.val[0] + 1 : tmp_pwend.val[0];

      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          auto ksize_w_divmod = divmods.ksize_w.Divmod(input_width);
          auto ksize_h_divmod = divmods.ksize_h.Divmod(input_height);
          auto tmp_width = ksize_w_divmod.val[1] > 0 ? ksize_w_divmod.val[0] + 1
                                                     : ksize_w_divmod.val[0];
          auto tmp_height = ksize_h_divmod.val[1] > 0
                                ? ksize_h_divmod.val[0] + 1
                                : ksize_h_divmod.val[0];
          int pool_size = tmp_height * tmp_width;
          int tmp_idx = ph * output_width + pw;
          int output_sub_idx =
              channel_last ? tmp_idx * divmods.channel.divisor + c_offset
                           : tmp_idx;
          T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                             : static_cast<T>(0);
          pool_process.compute(input, ouput_value, output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size),
                               &input_grad_data);
        }
      }
    } else {
      auto stride_height_div = divmods.stride_h.Div(h_offset - ksize_height);
      auto stride_width_div = divmods.stride_w.Div(w_offset - ksize_width);
      phstart = (h_offset < ksize_height) ? 0 : stride_height_div + 1;
      pwstart = (w_offset < ksize_width) ? 0 : stride_width_div + 1;
      phend = min(divmods.stride_h.Div(h_offset) + 1, output_height);
      pwend = min(divmods.stride_w.Div(w_offset) + 1, output_width);

      if (exclusive) {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            int hstart = ph * stride_height - padding_height;
            int wstart = pw * stride_width - padding_width;
            int hend = min(hstart + ksize_height, input_height);
            int wend = min(wstart + ksize_width, input_width);
            hstart = max(hstart, 0);
            wstart = max(wstart, 0);
            int pool_size = (hend - hstart) * (wend - wstart);
            int tmp_idx = ph * output_width + pw;
            int output_sub_idx =
                channel_last ? tmp_idx * divmods.channel.divisor + c_offset
                             : tmp_idx;
            T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                               : static_cast<T>(0);
            pool_process.compute(
                input, ouput_value, output_grad[output_sub_idx],
                static_cast<T>(1.0 / pool_size), &input_grad_data);
          }
        }
      } else {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            int pool_size = ksize_height * ksize_width;
            int tmp_idx = ph * output_width + pw;
            int output_sub_idx =
                channel_last ? tmp_idx * divmods.channel.divisor + c_offset
                             : tmp_idx;
            T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                               : static_cast<T>(0);
            pool_process.compute(
                input, ouput_value, output_grad[output_sub_idx],
                static_cast<T>(1.0 / pool_size), &input_grad_data);
          }
        }
      }
    }
    input_grad[index] = input_grad_data;
  }
}

template <typename T>
__global__ void KernelMaxPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    T* input_grad, FastDivModForPooling divmods, bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset, h_offset, c_offset, input_offset;
    OffsetPreparationFor4Dimension<FastDivModForPooling>(
        index, channel_last, divmods, 0, 0, input_width, input_height,
        &w_offset, &h_offset, &c_offset, &input_offset);
    input_data += input_offset;
    input_grad += input_offset;

    int hstart = h_offset * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = w_offset * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        int input_data_idx = channel_last
                                 ? (h * input_width + w) * channels + c_offset
                                 : h * input_width + w;
        if (ele == input_data[input_data_idx]) {
          maxIndex = input_data_idx;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIndex, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
void Pool2dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input, const std::vector<int>& input_shape,
    const std::vector<int>& output_shape, const std::vector<int>& ksize,
    const std::vector<int>& strides, const std::vector<int>& paddings,
    bool exclusive, bool adaptive, T* output, gpuStream_t stream,
    PoolProcess pool_compute) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_height = input_shape[2];
  const int input_width = input_shape[3];
  const int output_channels = output_shape[1];
  const int output_height = output_shape[2];
  const int output_width = output_shape[3];
  const int ksize_height = ksize[0];
  const int ksize_width = ksize[1];
  const int stride_height = strides[0];
  const int stride_width = strides[1];
  const int padding_height = paddings[0];
  const int padding_width = paddings[1];

  int nthreads = batch_size * output_channels * output_height * output_width;
  int thread_num = 1024;
#ifdef WITH_NV_JETSON
  // platform::ChangeThreadNum(context, &thread_num);
  thread_num = 512;
#endif
  int blocks = (nthreads + thread_num - 1) / thread_num;
  dim3 threads(thread_num, 1);
  dim3 grid(blocks, 1);

  auto pool_divmods =
      FastDivModForPooling(input_channels, output_width, output_height);
  KernelPool2D<PoolProcess, T><<<grid, threads, 0, stream>>>(
      nthreads, input, input_channels, input_height, input_width, output_height,
      output_width, ksize_height, ksize_width, stride_height, stride_width,
      padding_height, padding_width, pool_divmods, pool_compute, exclusive,
      adaptive, output);
}

/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* output,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    platform::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelPool2D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_divmods, pool_process,
        exclusive, adaptive, output_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* output, PoolProcess pool_process) {
    bool channel_last = (data_format == "NHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output->dims()[3] : output->dims()[1];
    const int output_height =
        channel_last ? output->dims()[1] : output->dims()[2];
    const int output_width =
        channel_last ? output->dims()[2] : output->dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    platform::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelPool2D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_divmods, pool_process,
        exclusive, adaptive, output_data, channel_last);
  }
};
/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* input_grad,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    auto pool_divmods = FastDivModForPoolingWithMoreStaff(
        input_channels, input_width, input_height, ksize_width, ksize_height,
        stride_width, stride_height);

    auto config = GetGpuLaunchConfig1D(context, nthreads);
    KernelPool2DGrad<T, PoolProcess><<<
        config.block_per_grid, config.thread_per_block, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, output_width,
        output_height, input_width, input_height, ksize_width, ksize_height,
        stride_width, stride_height, padding_width, padding_height,
        pool_divmods, pool_process, exclusive, adaptive, input_grad_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* input_grad, PoolProcess pool_process) {
    bool channel_last = (data_format == "NHWC");

    const int batch_size = input.dims()[0];
    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output.dims()[3] : output.dims()[1];
    const int output_height =
        channel_last ? output.dims()[1] : output.dims()[2];
    const int output_width = channel_last ? output.dims()[2] : output.dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    auto pool_divmods = FastDivModForPoolingWithMoreStaff(
        input_channels, input_width, input_height, ksize_width, ksize_height,
        stride_width, stride_height);

    auto config = GetGpuLaunchConfig1D(context, nthreads);
    KernelPool2DGrad<T, PoolProcess><<<
        config.block_per_grid, config.thread_per_block, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, output_width,
        output_height, input_width, input_height, ksize_width, ksize_height,
        stride_width, stride_height, padding_width, padding_height,
        pool_divmods, pool_process, exclusive, adaptive, input_grad_data,
        channel_last);
  }
};

/*
 * Tensors are in NCHW or NHWC format.
 * Ksize, strides are two elements. These two elements represent height
 * and width, respectively.
 * Paddings are four elements. These four elements represent height_up,
 * height_down, width_left and width_right, respectively.
 */
template <typename T>
class MaxPool2dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelMaxPool2DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        input_grad_data, pool_divmods);
  }
  void operator()(
      const platform::CUDADeviceContext& context,
      const framework::Tensor& input, const framework::Tensor& output,
      const framework::Tensor& output_grad, const std::vector<int>& ksize,
      const std::vector<int>& strides, const std::vector<int>& paddings,
      const std::string data_format, framework::Tensor* input_grad) {
    bool channel_last = (data_format == "NHWC");

    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[3] : input.dims()[1];
    const int input_height = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_width = channel_last ? input.dims()[2] : input.dims()[3];

    const int output_channels =
        channel_last ? output.dims()[3] : output.dims()[1];
    const int output_height =
        channel_last ? output.dims()[1] : output.dims()[2];
    const int output_width = channel_last ? output.dims()[2] : output.dims()[3];

    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];

    const int stride_height = strides[0];
    const int stride_width = strides[1];

    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);

    KernelMaxPool2DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        input_grad_data, pool_divmods, channel_last);
  }
};

template class Pool2dDirectCUDAFunctor<paddle::operators::math::MaxPool<float>,
                                       float>;
template class Pool2dDirectCUDAFunctor<paddle::operators::math::AvgPool<float>,
                                       float>;

template class MaxPool2dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool2dGradFunctor<platform::CUDADeviceContext, double>;
template class MaxPool2dGradFunctor<platform::CUDADeviceContext,
                                    paddle::platform::float16>;

template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template class Pool2dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool2dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool2dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool2dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3D(
    const int nthreads, const T* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, bool adaptive, T* output_data,
    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, pd, c, batch_idx;
    if (!channel_last) {
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      pd = (index / output_width / output_height) % output_depth;
      c = (index / output_width / output_height / output_depth) % channels;
      batch_idx =
          index / output_width / output_height / output_depth / channels;
    } else {
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      pd = (index / channels / output_width / output_height) % output_depth;
      batch_idx =
          index / channels / output_width / output_height / output_depth;
    }

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = AdaptStartIndex(pd, input_depth, output_depth);
      dend = AdaptEndIndex(pd, input_depth, output_depth);

      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    int input_data_stride;
    if (!channel_last) { /* NCDHW */
      input_data_stride =
          (batch_idx * channels + c) * input_depth * input_height * input_width;
    } else { /* NDHWC */
      input_data_stride =
          batch_idx * input_depth * input_height * input_width * channels;
    }
    input_data += input_data_stride;

    T ele = pool_process.initial();
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          auto input_data_idx =
              channel_last
                  ? ((d * input_height + h) * input_width + w) * channels + c
                  : (d * input_height + h) * input_width + w;
          pool_process.compute(input_data[input_data_idx], &ele);
        }
      }
    }
    int pool_size = (exclusive || adaptive)
                        ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                        : ksize_depth * ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename T, typename PoolProcess>
__global__ void KernelPool3DGrad(
    const int nthreads, const T* __restrict__ input_data,
    const T* __restrict__ output_data, const T* __restrict__ output_grad,
    const int channels, const int input_depth, const int input_height,
    const int input_width, const int output_depth, const int output_height,
    const int output_width, const int ksize_depth, const int ksize_height,
    const int ksize_width, const int stride_depth, const int stride_height,
    const int stride_width, const int padding_depth, const int padding_height,
    const int padding_width, PoolProcess pool_process, bool exclusive,
    bool adaptive, T* input_grad, bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset, h_offset, d_offset, c_offset, batch_idx, output_stride;
    T input = static_cast<T>(0);
    if (!channel_last) { /* "NCDHW" */
      w_offset = index % input_width + padding_width;
      h_offset = (index / input_width) % input_height + padding_height;
      d_offset =
          (index / input_width / input_height) % input_depth + padding_depth;
      c_offset = (index / input_width / input_height / input_depth) % channels;
      batch_idx = index / input_width / input_height / input_depth / channels;
      output_stride = (batch_idx * channels + c_offset) * output_depth *
                      output_height * output_width;
    } else { /* "NDHWC" */
      c_offset = index % channels;
      w_offset = (index / channels) % input_width + padding_width;
      h_offset =
          (index / channels / input_width) % input_height + padding_height;
      d_offset = (index / channels / input_width / input_height) % input_depth +
                 padding_depth;
      batch_idx = index / channels / input_width / input_height / input_depth;
      output_stride =
          batch_idx * output_depth * output_height * output_width * channels;
    }

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = AdaptStartIndex(d_offset, output_depth, input_depth);
      pdend = AdaptEndIndex(d_offset, output_depth, input_depth);

      phstart = AdaptStartIndex(h_offset, output_height, input_height);
      phend = AdaptEndIndex(h_offset, output_height, input_height);

      pwstart = AdaptStartIndex(w_offset, output_width, input_width);
      pwend = AdaptEndIndex(w_offset, output_width, input_width);
    } else {
      pdstart = (d_offset < ksize_depth)
                    ? 0
                    : (d_offset - ksize_depth) / stride_depth + 1;
      phstart = (h_offset < ksize_height)
                    ? 0
                    : (h_offset - ksize_height) / stride_height + 1;
      pwstart = (w_offset < ksize_width)
                    ? 0
                    : (w_offset - ksize_width) / stride_width + 1;
      pdend = min((d_offset) / stride_depth + 1, output_depth);
      phend = min((h_offset) / stride_height + 1, output_height);
      pwend = min((w_offset) / stride_width + 1, output_width);
    }
    if (pool_process.use_x) {
      input = input_data[index];
      output_data += output_stride;
    }
    output_grad += output_stride;
    T input_grad_data = static_cast<T>(0.0);

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int pool_size;
          if (adaptive) {
            pool_size =
                static_cast<int>(
                    ceil(static_cast<double>(input_depth) / ksize_depth)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_height) / ksize_height)) *
                static_cast<int>(
                    ceil(static_cast<double>(input_width) / ksize_width));
          } else {
            int dstart = pd * stride_depth - padding_depth;
            int hstart = ph * stride_height - padding_height;
            int wstart = pw * stride_width - padding_width;
            int dend = min(dstart + ksize_depth, input_depth);
            int hend = min(hstart + ksize_height, input_height);
            int wend = min(wstart + ksize_width, input_width);
            dstart = max(dstart, 0);
            hstart = max(hstart, 0);
            wstart = max(wstart, 0);
            pool_size =
                exclusive ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                          : ksize_depth * ksize_height * ksize_width;
          }

          int output_sub_idx =
              channel_last
                  ? ((pd * output_height + ph) * output_width + pw) * channels +
                        c_offset
                  : (pd * output_height + ph) * output_width + pw;
          T ouput_value = pool_process.use_x ? output_data[output_sub_idx]
                                             : static_cast<T>(0);
          pool_process.compute(input, ouput_value, output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size),
                               &input_grad_data);
        }
      }
    }
    input_grad[index] = input_grad_data;
  }
}

template <typename T>
__global__ void KernelMaxPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, T* input_grad,
    bool channel_last = false) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw, ph, pd, c, batch_idx;

    if (!channel_last) { /*NCDHW*/
      pw = index % output_width;
      ph = (index / output_width) % output_height;
      pd = (index / output_width / output_height) % output_depth;
      c = (index / output_width / output_height / output_depth) % channels;
      batch_idx =
          index / output_width / output_height / output_depth / channels;
    } else { /*NDHWC*/
      c = index % channels;
      pw = (index / channels) % output_width;
      ph = (index / channels / output_width) % output_height;
      pd = (index / channels / output_width / output_height) % output_depth;
      batch_idx =
          index / channels / output_width / output_height / output_depth;
    }

    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;

    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);

    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);

    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;

    int input_stride;
    if (!channel_last) {
      input_stride =
          (batch_idx * channels + c) * input_depth * input_height * input_width;
    } else {
      input_stride =
          batch_idx * input_depth * input_height * input_width * channels;
    }
    input_data += input_stride;
    input_grad += input_stride;
    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          int input_data_idx =
              channel_last
                  ? ((d * input_height + h) * input_width + w) * channels + c
                  : (d * input_height + h) * input_width + w;
          if (ele == input_data[input_data_idx]) {
            stop = true;
            maxIdx = input_data_idx;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
void Pool3dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input, const std::vector<int>& input_shape,
    const std::vector<int>& output_shape, const std::vector<int>& ksize,
    const std::vector<int>& strides, const std::vector<int>& paddings,
    bool exclusive, bool adaptive, T* output, gpuStream_t stream,
    PoolProcess pool_compute) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_depth = input_shape[2];
  const int input_height = input_shape[3];
  const int input_width = input_shape[4];
  const int output_channels = output_shape[1];
  const int output_depth = output_shape[2];
  const int output_height = output_shape[3];
  const int output_width = output_shape[4];
  const int ksize_depth = ksize[0];
  const int ksize_height = ksize[1];
  const int ksize_width = ksize[2];
  const int stride_depth = strides[0];
  const int stride_height = strides[1];
  const int stride_width = strides[2];
  const int padding_depth = paddings[0];
  const int padding_height = paddings[1];
  const int padding_width = paddings[2];

  int nthreads = batch_size * output_channels * output_depth * output_height *
                 output_width;
  int thread_num = 1024;
#ifdef WITH_NV_JETSON
  thread_num = 512;
#endif
  int blocks = (nthreads + thread_num - 1) / thread_num;
  dim3 threads(thread_num, 1);
  dim3 grid(blocks, 1);

  KernelPool3D<PoolProcess, T><<<grid, threads, 0, stream>>>(
      nthreads, input, input_channels, input_depth, input_height, input_width,
      output_depth, output_height, output_width, ksize_depth, ksize_height,
      ksize_width, stride_depth, stride_height, stride_width, padding_depth,
      padding_height, padding_width, pool_compute, exclusive, adaptive, output);
}

/*
 * Tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* output,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    platform::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* output, PoolProcess pool_process) {
    bool channel_last = (data_format == "NDHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output->dims()[4] : output->dims()[1];
    const int output_depth =
        channel_last ? output->dims()[1] : output->dims()[2];
    const int output_height =
        channel_last ? output->dims()[2] : output->dims()[3];
    const int output_width =
        channel_last ? output->dims()[3] : output->dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    platform::ChangeThreadNum(context, &thread_num);
#endif
    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data, channel_last);
  }
};

/*
 * Tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool exclusive,
                  bool adaptive, framework::Tensor* input_grad,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<T, PoolProcess><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, pool_process, exclusive, adaptive, input_grad_data);
  }
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  const std::string data_format, bool exclusive, bool adaptive,
                  framework::Tensor* input_grad, PoolProcess pool_process) {
    bool channel_last = (data_format == "NDHWC");

    const int batch_size = input.dims()[0];
    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output.dims()[4] : output.dims()[1];
    const int output_depth = channel_last ? output.dims()[1] : output.dims()[2];
    const int output_height =
        channel_last ? output.dims()[2] : output.dims()[3];
    const int output_width = channel_last ? output.dims()[3] : output.dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<T, PoolProcess><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, pool_process, exclusive, adaptive, input_grad_data,
        channel_last);  // add channel_last
  }
};

/*
 * tensors are in NCDHW or NDHWC format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 * Paddings are six elements. These six elements represent depth_forth,
 * depth_back,
 * height_up, height_down, width_left and width_right, respectively.
 */
template <class T>
class MaxPool3dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, input_grad_data);
  }
  void operator()(
      const platform::CUDADeviceContext& context,
      const framework::Tensor& input, const framework::Tensor& output,
      const framework::Tensor& output_grad, const std::vector<int>& ksize,
      const std::vector<int>& strides, const std::vector<int>& paddings,
      const std::string data_format, framework::Tensor* input_grad) {
    bool channel_last = (data_format == "NDHWC");
    const int batch_size = input.dims()[0];

    const int input_channels = channel_last ? input.dims()[4] : input.dims()[1];
    const int input_depth = channel_last ? input.dims()[1] : input.dims()[2];
    const int input_height = channel_last ? input.dims()[2] : input.dims()[3];
    const int input_width = channel_last ? input.dims()[3] : input.dims()[4];

    const int output_channels =
        channel_last ? output.dims()[4] : output.dims()[1];
    const int output_depth = channel_last ? output.dims()[1] : output.dims()[2];
    const int output_height =
        channel_last ? output.dims()[2] : output.dims()[3];
    const int output_width = channel_last ? output.dims()[3] : output.dims()[4];

    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];

    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];

    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, input_grad_data, channel_last);  // add channel_last
  }
};

template class Pool3dDirectCUDAFunctor<paddle::operators::math::MaxPool<float>,
                                       float>;
template class Pool3dDirectCUDAFunctor<paddle::operators::math::AvgPool<float>,
                                       float>;

template class MaxPool3dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool3dGradFunctor<platform::CUDADeviceContext, double>;
template class MaxPool3dGradFunctor<platform::CUDADeviceContext,
                                    paddle::platform::float16>;

template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template class Pool3dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool3dFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPool<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool3dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::MaxPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;
template class Pool3dGradFunctor<
    platform::CUDADeviceContext,
    paddle::operators::math::AvgPoolGrad<paddle::platform::float16>,
    paddle::platform::float16>;

template <typename T1, typename T2>
__global__ void KernelMaxPool2dWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, bool adaptive, T1* output_data, T2* mask_data,
    FastDivModForPooling divmods) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int hstart, hend, wstart, wend;
    int w_offset, h_offset, c_offset, input_offset;
    OffsetPreparationFor4Dimension<FastDivModForPooling>(
        index, false, divmods, 0, 0, input_width, input_height, &w_offset,
        &h_offset, &c_offset, &input_offset);
    input_data += input_offset;

    if (adaptive) {
      hstart = AdaptStartIndex(h_offset, input_height, output_height);
      hend = AdaptEndIndex(h_offset, input_height, output_height);

      wstart = AdaptStartIndex(w_offset, input_width, output_width);
      wend = AdaptEndIndex(w_offset, input_width, output_width);
    } else {
      hstart = h_offset * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = w_offset * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    T1 ele = -FLT_MAX;
    int max_index = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_index = h * input_width + w;
        if (ele < input_data[input_index]) {
          max_index = input_index;
          ele = input_data[input_index];
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool2DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask_data,
    const int channels, const int input_height, const int input_width,
    const int output_height, const int output_width, const int ksize_height,
    const int ksize_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, bool adaptive,
    T1* input_grad, FastDivModForPooling divmods) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int phstart, phend, pwstart, pwend;
    int w_offset, h_offset, c_offset, output_offset;
    OffsetPreparationFor4Dimension<FastDivModForPooling>(
        index, false, divmods, 0, 0, output_width, output_height, &w_offset,
        &h_offset, &c_offset, &output_offset);
    mask_data += output_offset;
    output_grad += output_offset;

    if (adaptive) {
      phstart = h_offset * output_height / input_height;
      phend =
          min((h_offset + 1) * output_height / input_height + 1, output_height);
      pwstart = w_offset * output_width / input_width;
      pwend =
          min((w_offset + 1) * output_width / input_width + 1, output_width);
    } else {
      phstart =
          (h_offset + padding_height < ksize_height)
              ? 0
              : (h_offset + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (w_offset + padding_width < ksize_width)
              ? 0
              : (w_offset + padding_width - ksize_width) / stride_width + 1;
      phend =
          min((h_offset + padding_height) / stride_height + 1, output_height);
      pwend = min((w_offset + padding_width) / stride_width + 1, output_width);
    }

    T1 input_grad_data = 0;
    int input_current_featuremap_idx = h_offset * input_width + w_offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_data[ph * output_width + pw] == input_current_featuremap_idx)
          input_grad_data += output_grad[ph * output_width + pw];
      }
    }
    input_grad[index] = input_grad_data;
  }
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* output, framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    platform::ChangeThreadNum(context, &thread_num);
#endif

    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, output_width, output_height);
    KernelMaxPool2dWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, adaptive, output_data,
        mask_data, pool_divmods);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_height = input_grad->dims()[2];
    const int input_width = input_grad->dims()[3];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T2* mask_data = mask.data<T2>();
    const T1* output_grad_data = output_grad.data<T1>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    auto pool_divmods =
        FastDivModForPooling(input_channels, input_width, input_height);
    KernelMaxPool2DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_height,
        input_width, output_height, output_width, ksize_height, ksize_width,
        stride_height, stride_width, padding_height, padding_width, adaptive,
        input_grad_data, pool_divmods);
  }
};

template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    bool adaptive, T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;

    int dstart, dend;
    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      dstart = AdaptStartIndex(pd, input_depth, output_depth);
      dend = AdaptEndIndex(pd, input_depth, output_depth);

      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      dstart = pd * stride_depth - padding_depth;
      hstart = ph * stride_height - padding_height;
      wstart = pw * stride_width - padding_width;
      dend = min(dstart + ksize_depth, input_depth);
      hend = min(hstart + ksize_height, input_height);
      wend = min(wstart + ksize_width, input_width);
      dstart = max(dstart, 0);
      hstart = max(hstart, 0);
      wstart = max(wstart, 0);
    }

    T1 ele = -FLT_MAX;
    int max_index = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (ele < input_data[(d * input_height + h) * input_width + w]) {
            max_index = (d * input_height + h) * input_width + w;
            ele = input_data[max_index];
          }
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask,
    const int channels, const int input_depth, const int input_height,
    const int input_width, const int output_depth, const int output_height,
    const int output_width, const int ksize_depth, const int ksize_height,
    const int ksize_width, const int stride_depth, const int stride_height,
    const int stride_width, const int padding_depth, const int padding_height,
    const int padding_width, bool adaptive, T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset = index % input_width;
    int h_offset = (index / input_width) % input_height;
    int d_offset = (index / input_width / input_height) % input_depth;
    int c_offset =
        (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart, pdend;
    int phstart, phend;
    int pwstart, pwend;
    if (adaptive) {
      pdstart = d_offset * output_depth / input_depth;
      pdend =
          min((d_offset + 1) * output_depth / input_depth + 1, output_depth);
      phstart = h_offset * output_height / input_height;
      phend =
          min((h_offset + 1) * output_height / input_height + 1, output_height);
      pwstart = w_offset * output_width / input_width;
      pwend =
          min((w_offset + 1) * output_width / input_width + 1, output_width);
    } else {
      pdstart =
          (d_offset + padding_depth < ksize_depth)
              ? 0
              : (d_offset + padding_depth - ksize_depth) / stride_depth + 1;
      phstart =
          (h_offset + padding_height < ksize_height)
              ? 0
              : (h_offset + padding_height - ksize_height) / stride_height + 1;
      pwstart =
          (w_offset + padding_width < ksize_width)
              ? 0
              : (w_offset + padding_width - ksize_width) / stride_width + 1;
      pdend = min((d_offset + padding_depth) / stride_depth + 1, output_depth);
      phend =
          min((h_offset + padding_height) / stride_height + 1, output_height);
      pwend = min((w_offset + padding_width) / stride_width + 1, output_width);
    }

    T1 input_grad_data = 0;
    int input_current_feature_map_idx =
        (d_offset * input_height + h_offset) * input_width + w_offset;
    int output_idx = (batch_idx * channels + c_offset) * output_depth *
                     output_height * output_width;
    mask += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          if (mask[(pd * output_height + ph) * output_width + pw] ==
              input_current_feature_map_idx)
            input_grad_data +=
                output_grad[(pd * output_height + ph) * output_width + pw];
        }
      }
    }
    input_grad[index] = input_grad_data;
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* output, framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int thread_num = 1024;
#ifdef WITH_NV_JETSON
    platform::ChangeThreadNum(context, &thread_num);
#endif

    int blocks = (nthreads + thread_num - 1) / thread_num;
    dim3 threads(thread_num, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, adaptive, output_data,
        mask_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, bool adaptive,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_depth = input_grad->dims()[2];
    const int input_height = input_grad->dims()[3];
    const int input_width = input_grad->dims()[4];
    const int output_depth = output_grad.dims()[2];
    const int output_height = output_grad.dims()[3];
    const int output_width = output_grad.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* output_grad_data = output_grad.data<T1>();
    const T2* mask_data = mask.data<T2>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_depth,
        input_height, input_width, output_depth, output_height, output_width,
        ksize_depth, ksize_height, ksize_width, stride_depth, stride_height,
        stride_width, padding_depth, padding_height, padding_width, adaptive,
        input_grad_data);
  }
};

template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
