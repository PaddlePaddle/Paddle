#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "paddle/fluid/operators/math/pooling.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(const int nthreads, const T* input_data,
                             const int channels, const int input_height,
                             const int input_width, const int output_height,
                             const int output_width, const int ksize_height,
                             const int ksize_width, const int stride_height,
                             const int stride_width, const int padding_height,
                             const int padding_width, PoolProcess pool_process,
                             bool exclusive, T* output_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        pool_process.compute(input_data[h * input_width + w], &ele);
      }
    }
    int pool_size = exclusive ? (hend - hstart) * (wend - wstart)
                              : ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetC = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int phstart = (offsetH < ksize_height)
                      ? 0
                      : (offsetH - ksize_height) / stride_height + 1;
    int pwstart = (offsetW < ksize_width)
                      ? 0
                      : (offsetW - ksize_width) / stride_width + 1;
    int phend = min(offsetH / stride_height + 1, output_height);
    int pwend = min(offsetW / stride_width + 1, output_width);
    T gradient = 0;
    T input = input_data[index];
    int output_idx =
        (batch_idx * channels + offsetC) * output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int hstart = ph * stride_height - padding_height;
        int wstart = pw * stride_width - padding_width;
        int hend = min(hstart + ksize_height, input_height);
        int wend = min(wstart + ksize_width, input_width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = exclusive ? (hend - hstart) * (wend - wstart)
                                  : ksize_height * ksize_width;
        int output_sub_idx = ph * output_width + pw;
        pool_process.compute(input, output_data[output_sub_idx],
                             output_grad[output_sub_idx],
                             static_cast<T>(1.0 / pool_size), &gradient);
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_height,
    const int input_width, const int output_height, const int output_width,
    const int ksize_height, const int ksize_width, const int stride_height,
    const int stride_width, const int padding_height, const int padding_width,
    T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    input_grad += (batch_idx * channels + c) * input_height * input_width;

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        if (ele == input_data[h * input_width + w]) {
          maxIndex = h * input_width + w;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIndex, output_grad[index]);
    }
  }
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_process, exclusive,
        output_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        pool_process, exclusive, input_grad_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T>
class MaxPool2dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_height, input_width, output_height, output_width, ksize_height,
        ksize_width, stride_height, stride_width, padding_height, padding_width,
        input_grad_data);
  }
};

template class MaxPool2dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool2dGradFunctor<platform::CUDADeviceContext, double>;

template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool2dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool2dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3D(
    const int nthreads, const T* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_process, bool exclusive, T* output_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = pool_process.initial();
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          pool_process.compute(
              input_data[(d * input_height + h) * input_width + w], &ele);
        }
      }
    }
    int pool_size = exclusive
                        ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                        : ksize_depth * ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, PoolProcess pool_process,
    bool exclusive, T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetD =
        (index / input_width / input_height) % input_depth + padding_depth;
    int offsetC = (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart = (offsetD < ksize_depth)
                      ? 0
                      : (offsetD - ksize_depth) / stride_depth + 1;
    int phstart = (offsetH < ksize_height)
                      ? 0
                      : (offsetH - ksize_height) / stride_height + 1;
    int pwstart = (offsetW < ksize_width)
                      ? 0
                      : (offsetW - ksize_width) / stride_width + 1;
    int pdend = min((offsetD) / stride_depth + 1, output_depth);
    int phend = min((offsetH) / stride_height + 1, output_height);
    int pwend = min((offsetW) / stride_width + 1, output_width);

    T gradient = 0;
    T input = input_data[index];
    int output_idx = (batch_idx * channels + offsetC) * output_depth *
                     output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int dstart = pd * stride_depth - padding_depth;
          int hstart = ph * stride_height - padding_height;
          int wstart = pw * stride_width - padding_width;
          int dend = min(dstart + ksize_depth, input_depth);
          int hend = min(hstart + ksize_height, input_height);
          int wend = min(wstart + ksize_width, input_width);
          dstart = max(dstart, 0);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          int pool_size =
              exclusive ? (dend - dstart) * (hend - hstart) * (wend - wstart)
                        : ksize_depth * ksize_height * ksize_width;
          int output_sub_idx = (pd * output_height + ph) * output_width + pw;
          pool_process.compute(input, output_data[output_sub_idx],
                               output_grad[output_sub_idx],
                               static_cast<T>(1.0 / pool_size), &gradient);
        }
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, const int channels, const int input_depth,
    const int input_height, const int input_width, const int output_depth,
    const int output_height, const int output_width, const int ksize_depth,
    const int ksize_height, const int ksize_width, const int stride_depth,
    const int stride_height, const int stride_width, const int padding_depth,
    const int padding_height, const int padding_width, T* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    input_grad +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          if (ele == input_data[(d * input_height + h) * input_width + w]) {
            stop = true;
            maxIdx = (d * input_height + h) * input_width + w;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      platform::CudaAtomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, framework::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, pool_process, exclusive,
        output_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename PoolProcess, class T>
class Pool3dGradFunctor<platform::CUDADeviceContext, PoolProcess, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<PoolProcess, T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, pool_process, exclusive, input_grad_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <class T>
class MaxPool3dGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad,
                  const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_channels,
        input_depth, input_height, input_width, output_depth, output_height,
        output_width, ksize_depth, ksize_height, ksize_width, stride_depth,
        stride_height, stride_width, padding_depth, padding_height,
        padding_width, input_grad_data);
  }
};

template class MaxPool3dGradFunctor<platform::CUDADeviceContext, float>;
template class MaxPool3dGradFunctor<platform::CUDADeviceContext, double>;

template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<float>,
                                 float>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<float>,
                                 float>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool3dFunctor<platform::CUDADeviceContext,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::MaxPoolGrad<double>,
                                 double>;
template class Pool3dGradFunctor<platform::CUDADeviceContext,
                                 paddle::operators::math::AvgPoolGrad<double>,
                                 double>;

template <typename T1, typename T2>
__global__ void KernelMaxPool2dWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T1 ele = -FLT_MAX;
    int max_index = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_index = h * input_width + w;
        if (ele < input_data[input_index]) {
          max_index = input_index;
          ele = input_data[input_index];
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool2DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask_data,
    const int channels, const int input_height, const int input_width,
    const int output_height, const int output_width, const int ksize_height,
    const int ksize_width, const int stride_height, const int stride_width,
    const int padding_height, const int padding_width, T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset = index % input_width;
    int h_offset = (index / input_width) % input_height;
    int c_offset = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int ph_start =
        (h_offset + padding_height < ksize_height)
            ? 0
            : (h_offset + padding_height - ksize_height) / stride_height + 1;
    int pw_start =
        (w_offset + padding_width < ksize_width)
            ? 0
            : (w_offset + padding_width - ksize_width) / stride_width + 1;
    int ph_end =
        min((h_offset + padding_height) / stride_height + 1, output_height);
    int pw_end =
        min((w_offset + padding_width) / stride_width + 1, output_width);

    T1 gradient = 0;
    int input_current_featuremap_idx = h_offset * input_width + w_offset;
    int output_idx =
        (batch_idx * channels + c_offset) * output_height * output_width;

    mask_data += output_idx;
    output_grad += output_idx;
    for (int ph = ph_start; ph < ph_end; ++ph) {
      for (int pw = pw_start; pw < pw_end; ++pw) {
        if (mask_data[ph * output_width + pw] == input_current_featuremap_idx)
          gradient += output_grad[ph * output_width + pw];
      }
    }
    input_grad[index] = gradient;
  }
}

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, framework::Tensor* output,
                  framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2dWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, output_data, mask_data);
  }
};

/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_height = input_grad->dims()[2];
    const int input_width = input_grad->dims()[3];
    const int output_height = output_grad.dims()[2];
    const int output_width = output_grad.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T2* mask_data = mask.data<T2>();
    const T1* output_grad_data = output_grad.data<T1>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_height,
        input_width, output_height, output_width, ksize_height, ksize_width,
        stride_height, stride_width, padding_height, padding_width,
        input_grad_data);
  }
};

template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool2dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool2dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdx(
    const int nthreads, const T1* input_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    T1* output_data, T2* mask_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;

    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);

    T1 ele = -FLT_MAX;
    int max_index = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          if (ele < input_data[(d * input_height + h) * input_width + w]) {
            max_index = (d * input_height + h) * input_width + w;
            ele = input_data[max_index];
          }
        }
      }
    }
    output_data[index] = ele;
    mask_data[index] = max_index;
  }
}

template <typename T1, typename T2>
__global__ void KernelMaxPool3DWithIdxGrad(
    const int nthreads, const T1* output_grad, const T2* mask,
    const int channels, const int input_depth, const int input_height,
    const int input_width, const int output_depth, const int output_height,
    const int output_width, const int ksize_depth, const int ksize_height,
    const int ksize_width, const int stride_depth, const int stride_height,
    const int stride_width, const int padding_depth, const int padding_height,
    const int padding_width, T1* input_grad) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int w_offset = index % input_width;
    int h_offset = (index / input_width) % input_height;
    int d_offset = (index / input_width / input_height) % input_depth;
    int c_offset =
        (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pd_start =
        (d_offset + padding_depth < ksize_depth)
            ? 0
            : (d_offset + padding_depth - ksize_depth) / stride_depth + 1;
    int ph_start =
        (h_offset + padding_height < ksize_height)
            ? 0
            : (h_offset + padding_height - ksize_height) / stride_height + 1;
    int pw_start =
        (w_offset + padding_width < ksize_width)
            ? 0
            : (w_offset + padding_width - ksize_width) / stride_width + 1;
    int pd_end =
        min((d_offset + padding_depth) / stride_depth + 1, output_depth);
    int ph_end =
        min((h_offset + padding_height) / stride_height + 1, output_height);
    int pw_end =
        min((w_offset + padding_width) / stride_width + 1, output_width);

    T1 gradient = 0;
    int input_current_feature_map_idx =
        (d_offset * input_height + h_offset) * input_width + w_offset;
    int output_idx = (batch_idx * channels + c_offset) * output_depth *
                     output_height * output_width;
    mask += output_idx;
    output_grad += output_idx;

    for (int pd = pd_start; pd < pd_end; ++pd) {
      for (int ph = ph_start; ph < ph_end; ++ph) {
        for (int pw = pw_start; pw < pw_end; ++pw) {
          if (mask[(pd * output_height + ph) * output_width + pw] ==
              input_current_feature_map_idx)
            gradient +=
                output_grad[(pd * output_height + ph) * output_width + pw];
        }
      }
    }
    input_grad[index] = gradient;
  }
}

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, framework::Tensor* output,
                  framework::Tensor* mask) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output->dims()[1];
    const int output_depth = output->dims()[2];
    const int output_height = output->dims()[3];
    const int output_width = output->dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* input_data = input.data<T1>();
    T1* output_data = output->mutable_data<T1>(context.GetPlace());
    T2* mask_data = mask->mutable_data<T2>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdx<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_depth, input_height,
        input_width, output_depth, output_height, output_width, ksize_depth,
        ksize_height, ksize_width, stride_depth, stride_height, stride_width,
        padding_depth, padding_height, padding_width, output_data, mask_data);
  }
};

/*
 * All tensors are in NCDHW format.
 * Ksize, strides, paddings are three elements. These three elements represent
 * depth, height and width, respectively.
 */
template <typename T1, typename T2>
class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, T1, T2> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& output_grad,
                  const framework::Tensor& mask, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* input_grad) {
    const int batch_size = input_grad->dims()[0];
    const int input_channels = input_grad->dims()[1];
    const int input_depth = input_grad->dims()[2];
    const int input_height = input_grad->dims()[3];
    const int input_width = input_grad->dims()[4];
    const int output_depth = output_grad.dims()[2];
    const int output_height = output_grad.dims()[3];
    const int output_width = output_grad.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T1* output_grad_data = output_grad.data<T1>();
    const T2* mask_data = mask.data<T2>();
    T1* input_grad_data = input_grad->mutable_data<T1>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DWithIdxGrad<T1, T2><<<grid, threads, 0, context.stream()>>>(
        nthreads, output_grad_data, mask_data, input_channels, input_depth,
        input_height, input_width, output_depth, output_height, output_width,
        ksize_depth, ksize_height, ksize_width, stride_depth, stride_height,
        stride_width, padding_depth, padding_height, padding_width,
        input_grad_data);
  }
};

template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, float,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext, float,
                                             int>;
template class MaxPool3dWithIndexFunctor<platform::CUDADeviceContext, double,
                                         int>;
template class MaxPool3dWithIndexGradFunctor<platform::CUDADeviceContext,
                                             double, int>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
