#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "paddle/fluid/operators/math/im2col.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <class T>
__global__ void im2col(const T* data_im, int num_outs, int im_height,
                       int im_width, int dilation_h, int dilation_w,
                       int filter_height, int filter_width, int stride_height,
                       int stride_width, int padding_height, int padding_width,
                       int col_height, int col_width, T* data_col) {
  const int index =
      (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  if (index < num_outs) {
    int w_out = index % col_width;
    int h_out = (index / col_width) % col_height;
    int channel_in = index / col_width / col_height;
    int channel_out = channel_in * filter_height * filter_width;
    int h_in = h_out * stride_height - padding_height;
    int w_in = w_out * stride_width - padding_width;

    data_col += (channel_out * col_height + h_out) * col_width + w_out;
    data_im += (channel_in * im_height + h_in) * im_width + w_in;
    for (int i = 0; i < filter_height; ++i) {
      for (int j = 0; j < filter_width; ++j) {
        int rIdx = h_in + i * dilation_h;
        int cIdx = w_in + j * dilation_w;
        *data_col =
            (rIdx >= im_height || rIdx < 0 || cIdx >= im_width || cIdx < 0)
                ? 0
                : data_im[i * dilation_h * im_width + j * dilation_w];
        data_col += col_height * col_width;
      }
    }
  }
}

/*
 * im = [input_channels, input_height, input_width]
 * col =
 *   [input_channels, filter_height, filter_width, output_height, output_width]
 */
template <class T>
class Im2ColFunctor<paddle::operators::math::ColFormat::kCFO,
                    platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& im, const std::vector<int>& dilation,
                  const std::vector<int>& stride,
                  const std::vector<int>& padding, framework::Tensor* col) {
    PADDLE_ENFORCE(im.dims().size() == 3);
    PADDLE_ENFORCE(col->dims().size() == 5);

    int im_channels = im.dims()[0];
    int im_height = im.dims()[1];
    int im_width = im.dims()[2];
    int filter_height = col->dims()[1];
    int filter_width = col->dims()[2];
    int col_height = col->dims()[3];
    int col_width = col->dims()[4];

    int num_outputs = im_channels * col_height * col_width;
    int blocks = (num_outputs + 1024 - 1) / 1024;
    int block_x = 512;
    int block_y = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(block_x, block_y);
    im2col<T><<<grid, threads, 0, context.stream()>>>(
        im.data<T>(), num_outputs, im_height, im_width, dilation[0],
        dilation[1], filter_height, filter_width, stride[0], stride[1],
        padding[0], padding[1], col_height, col_width, col->data<T>());
  }
};

template <class T>
__global__ void col2im(int n, const T* data_col, int im_height, int im_width,
                       int dilation_h, int dilation_w, int filter_height,
                       int filter_width, int stride_height, int stride_width,
                       int padding_height, int padding_width, int col_height,
                       int col_width, T* data_im) {
  const int index =
      (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  const int d_filter_height = dilation_h * (filter_height - 1) + 1;
  const int d_filter_width = dilation_w * (filter_width - 1) + 1;

  if (index < n) {
    T val = 0;
    int w = index % im_width + padding_width;
    int h = (index / im_width) % im_height + padding_height;
    int c = index / (im_width * im_height);

    // compute the start and end of the output
    int w_col_start =
        (w < d_filter_width) ? 0 : (w - d_filter_width) / stride_width + 1;
    int w_col_end = min(w / stride_width + 1, col_width);
    int h_col_start =
        (h < d_filter_height) ? 0 : (h - d_filter_height) / stride_height + 1;
    int h_col_end = min(h / stride_height + 1, col_height);

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_off = (h - h_col * stride_height);
        int w_off = (w - w_col * stride_width);
        if (h_off % dilation_h == 0 && w_off % dilation_w == 0) {
          h_off /= dilation_h;
          w_off /= dilation_w;
          int data_col_index =
              (((c * filter_height + h_off) * filter_width + w_off) *
                   col_height +
               h_col) *
                  col_width +
              w_col;

          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

/*
 * im = [input_channels, input_height, input_width]
 * col =
 *   [input_channels, filter_height, filter_width, output_height, output_width]
 */
template <class T>
class Col2ImFunctor<paddle::operators::math::ColFormat::kCFO,
                    platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& col,
                  const std::vector<int>& dilation,
                  const std::vector<int>& stride,
                  const std::vector<int>& padding, framework::Tensor* im) {
    PADDLE_ENFORCE(im->dims().size() == 3);
    PADDLE_ENFORCE(col.dims().size() == 5);

    int im_channels = im->dims()[0];
    int im_height = im->dims()[1];
    int im_width = im->dims()[2];
    int filter_height = col.dims()[1];
    int filter_width = col.dims()[2];
    int col_height = col.dims()[3];
    int col_width = col.dims()[4];

    PADDLE_ENFORCE_EQ((im_height + padding[0] + padding[2] -
                       (dilation[0] * (filter_height - 1) + 1)) /
                              stride[0] +
                          1,
                      col_height,
                      "Output_height and padding(padding_up, padding_down) are "
                      "inconsistent.");
    PADDLE_ENFORCE_EQ((im_width + padding[1] + padding[3] -
                       (dilation[1] * (filter_width - 1) + 1)) /
                              stride[1] +
                          1,
                      col_width,
                      "col_width and padding(padding_left, padding_right) are "
                      "inconsistent.");

    size_t num_kernels = im_channels * im_height * im_width;

    size_t blocks = (num_kernels + 1024 - 1) / 1024;
    size_t block_x = 512;
    size_t block_y = (blocks + 512 - 1) / 512;
    dim3 threads(1024, 1);
    dim3 grid(block_x, block_y);

    // To avoid involving atomic operations, we will launch one kernel per
    // bottom dimension, and then in the kernel add up the top dimensions.
    col2im<T><<<grid, threads, 0, context.stream()>>>(
        num_kernels, col.data<T>(), im_height, im_width, dilation[0],
        dilation[1], filter_height, filter_width, stride[0], stride[1],
        padding[0], padding[2], col_height, col_width, im->data<T>());
  }
};

template class Im2ColFunctor<paddle::operators::math::ColFormat::kCFO,
                             platform::CUDADeviceContext, float>;
template class Im2ColFunctor<paddle::operators::math::ColFormat::kCFO,
                             platform::CUDADeviceContext, double>;
template class Col2ImFunctor<paddle::operators::math::ColFormat::kCFO,
                             platform::CUDADeviceContext, float>;
template class Col2ImFunctor<paddle::operators::math::ColFormat::kCFO,
                             platform::CUDADeviceContext, double>;

template <class T>
__global__ void im2colOCF(const T* im_data, int im_channels, int im_height,
                          int im_width, int filter_height, int filter_width,
                          int stride_height, int stride_width,
                          int padding_height, int padding_width, int col_height,
                          int col_width, T* col_data) {
  int swid = blockIdx.x;
  int shid = blockIdx.y;
  for (int channelid = threadIdx.z; channelid < im_channels;
       channelid += blockDim.z) {
    for (int idy = threadIdx.y; idy < filter_height; idy += blockDim.y) {
      for (int idx = threadIdx.x; idx < filter_width; idx += blockDim.x) {
        int width_offset = idx + swid * stride_width - padding_width;
        int height_offset = idy + shid * stride_height - padding_height;
        int im_offset = width_offset + height_offset * im_width +
                        channelid * im_height * im_width;

        int col_offset = idx + idy * filter_width +
                         channelid * filter_height * filter_width +
                         (shid * col_width + swid) *
                             (im_channels * filter_height * filter_width);

        col_data[col_offset] =
            (height_offset >= im_height || height_offset < 0 ||
             width_offset >= im_width || width_offset < 0)
                ? T(0)
                : im_data[im_offset];
      }
    }
  }
}

/*
 * im = [input_channels, input_height, input_width]
 * col =
 *   [output_height, output_width, input_channels, filter_height, filter_width]
 */
template <class T>
class Im2ColFunctor<paddle::operators::math::ColFormat::kOCF,
                    platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& im, const std::vector<int>& dilation,
                  const std::vector<int>& stride,
                  const std::vector<int>& padding, framework::Tensor* col) {
    PADDLE_ENFORCE(im.dims().size() == 3);
    PADDLE_ENFORCE(col->dims().size() == 5);
    int im_channels = im.dims()[0];
    int im_height = im.dims()[1];
    int im_width = im.dims()[2];
    int filter_height = col->dims()[3];
    int filter_width = col->dims()[4];
    int col_height = col->dims()[0];
    int col_width = col->dims()[1];

    int block_dim_x = 0;
    int block_dim_y = 0;
    if (filter_height <= 4 && filter_width <= 4) {
      block_dim_x = 4;
      block_dim_y = 4;
    } else if (filter_height <= 8 && filter_width <= 8) {
      block_dim_x = 8;
      block_dim_y = 8;
    } else if (filter_height <= 16 && filter_width <= 16) {
      block_dim_x = 16;
      block_dim_y = 16;
    } else {
      block_dim_x = 32;
      block_dim_y = 32;
    }

    int block_dim_z = 1024 / block_dim_x / block_dim_y;
    dim3 threads(block_dim_x, block_dim_y, std::min(block_dim_z, im_channels));
    dim3 grid(col_width, col_height);
    im2colOCF<T><<<grid, threads, 0, context.stream()>>>(
        im.data<T>(), im_channels, im_height, im_width, filter_height,
        filter_width, stride[0], stride[1], padding[0], padding[1], col_height,
        col_width, col->data<T>());
  }
};

template <class T>
__global__ void col2imOCF(const T* col_data, int im_channels, int im_height,
                          int im_width, int filter_height, int filter_width,
                          int stride_height, int stride_width,
                          int padding_height, int padding_width, int col_height,
                          int col_width, T* im_data) {
  int swid = blockIdx.x;
  int shid = blockIdx.y;
  for (int channelid = threadIdx.z; channelid < im_channels;
       channelid += blockDim.z) {
    for (int idy = threadIdx.y; idy < filter_height; idy += blockDim.y) {
      for (int idx = threadIdx.x; idx < filter_width; idx += blockDim.x) {
        int width_offset = idx + swid * stride_width - padding_width;
        int height_offset = idy + shid * stride_height - padding_height;
        int im_offset = width_offset + height_offset * im_width +
                        channelid * im_height * im_width;

        int col_offset = idx + idy * filter_width +
                         channelid * filter_height * filter_width +
                         (shid * col_width + swid) *
                             (im_channels * filter_height * filter_width);

        if (height_offset >= 0 && height_offset < im_height &&
            width_offset >= 0 && width_offset < im_width) {
          paddle::platform::CudaAtomicAdd(im_data + im_offset,
                                          col_data[col_offset]);
        }
      }
    }
  }
}

/*
 * im = [input_channels, input_height, input_width]
 * col =
 *   [output_height, output_width, input_channels, filter_height, filter_width]
 */
template <class T>
class Col2ImFunctor<paddle::operators::math::ColFormat::kOCF,
                    platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& col,
                  const std::vector<int>& dilation,
                  const std::vector<int>& stride,
                  const std::vector<int>& padding, framework::Tensor* im) {
    PADDLE_ENFORCE(im->dims().size() == 3);
    PADDLE_ENFORCE(col.dims().size() == 5);
    int im_channels = im->dims()[0];
    int im_height = im->dims()[1];
    int im_width = im->dims()[2];
    int filter_height = col.dims()[3];
    int filter_width = col.dims()[4];
    int col_height = col.dims()[0];
    int col_width = col.dims()[1];

    PADDLE_ENFORCE_EQ((im_height + padding[0] + padding[2] -
                       (dilation[0] * (filter_height - 1) + 1)) /
                              stride[0] +
                          1,
                      col_height,
                      "Output_height and padding(padding_up, padding_down) are "
                      "inconsistent.");
    PADDLE_ENFORCE_EQ((im_width + padding[1] + padding[3] -
                       (dilation[1] * (filter_width - 1) + 1)) /
                              stride[1] +
                          1,
                      col_width,
                      "col_width and padding(padding_left, padding_right) are "
                      "inconsistent.");

    int block_dim_x = 0;
    int block_dim_y = 0;
    if (filter_height <= 4 && filter_width <= 4) {
      block_dim_x = 4;
      block_dim_y = 4;
    } else if (filter_height <= 8 && filter_width <= 8) {
      block_dim_x = 8;
      block_dim_y = 8;
    } else if (filter_height <= 16 && filter_width <= 16) {
      block_dim_x = 16;
      block_dim_y = 16;
    } else {
      block_dim_x = 32;
      block_dim_y = 32;
    }

    int block_dim_z = 1024 / block_dim_x / block_dim_y;
    dim3 threads(block_dim_x, block_dim_y, std::min(block_dim_z, im_channels));
    dim3 grid(col_width, col_height);
    col2imOCF<T><<<grid, threads, 0, context.stream()>>>(
        col.data<T>(), im_channels, im_height, im_width, filter_height,
        filter_width, stride[0], stride[1], padding[0], padding[1], col_height,
        col_width, im->data<T>());
  }
};

template class Im2ColFunctor<paddle::operators::math::ColFormat::kOCF,
                             platform::CUDADeviceContext, float>;
template class Im2ColFunctor<paddle::operators::math::ColFormat::kOCF,
                             platform::CUDADeviceContext, double>;
template class Col2ImFunctor<paddle::operators::math::ColFormat::kOCF,
                             platform::CUDADeviceContext, float>;
template class Col2ImFunctor<paddle::operators::math::ColFormat::kOCF,
                             platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
