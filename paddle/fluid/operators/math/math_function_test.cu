//  Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "gtest/gtest.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/device_context.h"

void fill_fp16_data(paddle::platform::float16* in_ptr, size_t size,
                    const std::vector<float>& data) {
  PADDLE_ENFORCE_EQ(size, data.size());
  for (size_t i = 0; i < data.size(); ++i) {
    in_ptr[i] = paddle::platform::float16(data[i]);
  }
}

template <typename T>
inline paddle::operators::math::BlasT<paddle::platform::CUDADeviceContext, T>
GetBlas(const paddle::platform::CUDADeviceContext& context) {
  return paddle::operators::math::GetBlas<paddle::platform::CUDADeviceContext,
                                          T>(context);
}

TEST(math_function, notrans_mul_trans_fp32) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor out_gpu;
  paddle::framework::Tensor out;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  float* input1_ptr = input1.mutable_data<float>({2, 3}, cpu_place);
  float arr[6] = {0, 1, 2, 3, 4, 5};
  memcpy(input1_ptr, arr, 6 * sizeof(float));

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input1, gpu_place, &input2_gpu);

  out_gpu.mutable_data<float>({2, 2}, gpu_place);
  GetBlas<float>(context).MatMul(input1_gpu, false, input2_gpu, true, 1,
                                 &out_gpu, 0);

  paddle::framework::TensorCopySync(out_gpu, cpu_place, &out);

  float* out_ptr = out.data<float>();
  context.Wait();
  EXPECT_EQ(out_ptr[0], 5);
  EXPECT_EQ(out_ptr[1], 14);
  EXPECT_EQ(out_ptr[2], 14);
  EXPECT_EQ(out_ptr[3], 50);
}

TEST(math_function, notrans_mul_trans_fp16) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor out_gpu;
  paddle::framework::Tensor out;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  // fp16 GEMM in cublas requires GPU compute capability >= 53
  if (context.GetComputeCapability() < 53) {
    return;
  }

  paddle::platform::float16* input1_ptr =
      input1.mutable_data<paddle::platform::float16>({2, 3}, cpu_place);
  fill_fp16_data(input1_ptr, input1.numel(), {0, 1, 2, 3, 4, 5});

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input1, gpu_place, &input2_gpu);

  out_gpu.mutable_data<paddle::platform::float16>({2, 2}, gpu_place);

  GetBlas<paddle::platform::float16>(context).MatMul(
      input1_gpu, false, input2_gpu, true, paddle::platform::float16(1),
      &out_gpu, paddle::platform::float16(0));

  paddle::framework::TensorCopySync(out_gpu, cpu_place, &out);

  paddle::platform::float16* out_ptr = out.data<paddle::platform::float16>();
  context.Wait();
  EXPECT_EQ(static_cast<float>(out_ptr[0]), 5);
  EXPECT_EQ(static_cast<float>(out_ptr[1]), 14);
  EXPECT_EQ(static_cast<float>(out_ptr[2]), 14);
  EXPECT_EQ(static_cast<float>(out_ptr[3]), 50);
}

TEST(math_function, trans_mul_notrans_fp32) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor out_gpu;
  paddle::framework::Tensor out;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  float* input1_ptr = input1.mutable_data<float>({2, 3}, cpu_place);
  float arr[6] = {0, 1, 2, 3, 4, 5};
  memcpy(input1_ptr, arr, 6 * sizeof(float));

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input1, gpu_place, &input2_gpu);

  out_gpu.mutable_data<float>({3, 3}, gpu_place);

  GetBlas<float>(context).MatMul(input1_gpu, true, input2_gpu, false, 1,
                                 &out_gpu, 0);

  paddle::framework::TensorCopySync(out_gpu, cpu_place, &out);

  float* out_ptr = out.data<float>();
  context.Wait();
  EXPECT_EQ(out_ptr[0], 9);
  EXPECT_EQ(out_ptr[1], 12);
  EXPECT_EQ(out_ptr[2], 15);
  EXPECT_EQ(out_ptr[3], 12);
  EXPECT_EQ(out_ptr[4], 17);
  EXPECT_EQ(out_ptr[5], 22);
  EXPECT_EQ(out_ptr[6], 15);
  EXPECT_EQ(out_ptr[7], 22);
  EXPECT_EQ(out_ptr[8], 29);
}

TEST(math_function, trans_mul_notrans_fp16) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor out_gpu;
  paddle::framework::Tensor out;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  // fp16 GEMM in cublas requires GPU compute capability >= 53
  if (context.GetComputeCapability() < 53) {
    return;
  }

  paddle::platform::float16* input1_ptr =
      input1.mutable_data<paddle::platform::float16>({2, 3}, cpu_place);
  fill_fp16_data(input1_ptr, input1.numel(), {0, 1, 2, 3, 4, 5});

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input1, gpu_place, &input2_gpu);

  out_gpu.mutable_data<paddle::platform::float16>({3, 3}, gpu_place);

  GetBlas<paddle::platform::float16>(context).MatMul(
      input1_gpu, true, input2_gpu, false, paddle::platform::float16(1),
      &out_gpu, paddle::platform::float16(0));

  paddle::framework::TensorCopySync(out_gpu, cpu_place, &out);

  paddle::platform::float16* out_ptr = out.data<paddle::platform::float16>();
  context.Wait();
  EXPECT_EQ(static_cast<float>(out_ptr[0]), 9);
  EXPECT_EQ(static_cast<float>(out_ptr[1]), 12);
  EXPECT_EQ(static_cast<float>(out_ptr[2]), 15);
  EXPECT_EQ(static_cast<float>(out_ptr[3]), 12);
  EXPECT_EQ(static_cast<float>(out_ptr[4]), 17);
  EXPECT_EQ(static_cast<float>(out_ptr[5]), 22);
  EXPECT_EQ(static_cast<float>(out_ptr[6]), 15);
  EXPECT_EQ(static_cast<float>(out_ptr[7]), 22);
  EXPECT_EQ(static_cast<float>(out_ptr[8]), 29);
}

TEST(math_function, gemm_notrans_cublas_fp32) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input2;
  paddle::framework::Tensor input3;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor input3_gpu;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  int m = 2;
  int n = 3;
  int k = 3;
  float* input1_ptr = input1.mutable_data<float>({2, 3}, cpu_place);
  float arr1[6] = {0, 1, 2, 3, 4, 5};
  memcpy(input1_ptr, arr1, 6 * sizeof(float));
  float* input2_ptr = input2.mutable_data<float>({3, 4}, cpu_place);
  float arr2[12] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11};
  memcpy(input2_ptr, arr2, 12 * sizeof(float));
  float* input3_ptr = input3.mutable_data<float>({2, 4}, cpu_place);
  float arr3[8] = {0, 1, 2, 3, 4, 5, 6, 7};
  memcpy(input3_ptr, arr3, 8 * sizeof(float));

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input2, gpu_place, &input2_gpu);
  paddle::framework::TensorCopySync(input3, gpu_place, &input3_gpu);
  float* a = input1_gpu.data<float>();
  float* b = input2_gpu.data<float>();
  float* c = input3_gpu.mutable_data<float>(gpu_place);

  GetBlas<float>(context).GEMM(false, false, m, n, k, 1, a, 3, b + 1, 4, 1,
                               c + 1, 4);

  paddle::framework::TensorCopySync(input3_gpu, cpu_place, &input3);

  // numpy code:
  // a = np.arange(6).reshape(2, 3)
  // b = np.arange(12).reshape(3, 4)[:, 1:]
  // c = np.arange(8).reshape(2, 4)[:, 1:]
  // out = np.arange(8).reshape(2, 4)
  // out[:, 1:] = np.dot(a, b) + c
  context.Wait();
  EXPECT_EQ(input3_ptr[0], 0);
  EXPECT_EQ(input3_ptr[1], 24);
  EXPECT_EQ(input3_ptr[2], 28);
  EXPECT_EQ(input3_ptr[3], 32);
  EXPECT_EQ(input3_ptr[4], 4);
  EXPECT_EQ(input3_ptr[5], 73);
  EXPECT_EQ(input3_ptr[6], 86);
  EXPECT_EQ(input3_ptr[7], 99);
}

TEST(math_function, gemm_notrans_cublas_fp16) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input2;
  paddle::framework::Tensor input3;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor input3_gpu;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  // fp16 GEMM in cublas requires GPU compute capability >= 53
  if (context.GetComputeCapability() < 53) {
    return;
  }

  int m = 2;
  int n = 3;
  int k = 3;
  paddle::platform::float16* input1_ptr =
      input1.mutable_data<paddle::platform::float16>({2, 3}, cpu_place);
  fill_fp16_data(input1_ptr, input1.numel(), {0, 1, 2, 3, 4, 5});
  paddle::platform::float16* input2_ptr =
      input2.mutable_data<paddle::platform::float16>({3, 4}, cpu_place);
  fill_fp16_data(input2_ptr, input2.numel(),
                 {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11});
  paddle::platform::float16* input3_ptr =
      input3.mutable_data<paddle::platform::float16>({2, 4}, cpu_place);
  fill_fp16_data(input3_ptr, input3.numel(), {0, 1, 2, 3, 4, 5, 6, 7});

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input2, gpu_place, &input2_gpu);
  paddle::framework::TensorCopySync(input3, gpu_place, &input3_gpu);
  paddle::platform::float16* a = input1_gpu.data<paddle::platform::float16>();
  paddle::platform::float16* b = input2_gpu.data<paddle::platform::float16>();
  paddle::platform::float16* c =
      input3_gpu.mutable_data<paddle::platform::float16>(gpu_place);

  GetBlas<paddle::platform::float16>(context).GEMM(
      false, false, m, n, k, static_cast<paddle::platform::float16>(1), a, 3,
      b + 1, 4, static_cast<paddle::platform::float16>(1), c + 1, 4);

  paddle::framework::TensorCopySync(input3_gpu, cpu_place, &input3);

  // numpy code:
  // a = np.arange(6).reshape(2, 3)
  // b = np.arange(12).reshape(3, 4)[:, 1:]
  // c = np.arange(8).reshape(2, 4)[:, 1:]
  // out = np.arange(8).reshape(2, 4)
  // out[:, 1:] = np.dot(a, b) + c
  context.Wait();
  EXPECT_EQ(static_cast<float>(input3_ptr[0]), 0);
  EXPECT_EQ(static_cast<float>(input3_ptr[1]), 24);
  EXPECT_EQ(static_cast<float>(input3_ptr[2]), 28);
  EXPECT_EQ(static_cast<float>(input3_ptr[3]), 32);
  EXPECT_EQ(static_cast<float>(input3_ptr[4]), 4);
  EXPECT_EQ(static_cast<float>(input3_ptr[5]), 73);
  EXPECT_EQ(static_cast<float>(input3_ptr[6]), 86);
  EXPECT_EQ(static_cast<float>(input3_ptr[7]), 99);
}

TEST(math_function, gemm_trans_cublas_fp32) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input2;
  paddle::framework::Tensor input3;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor input3_gpu;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  int m = 2;
  int n = 3;
  int k = 3;
  float* input1_ptr = input1.mutable_data<float>({2, 3}, cpu_place);
  float arr1[6] = {0, 1, 2, 3, 4, 5};
  memcpy(input1_ptr, arr1, 6 * sizeof(float));
  float* input2_ptr = input2.mutable_data<float>({4, 3}, cpu_place);
  float arr2[12] = {0, 4, 8, 1, 5, 9, 2, 6, 10, 3, 7, 11};
  memcpy(input2_ptr, arr2, 12 * sizeof(float));
  float* input3_ptr = input3.mutable_data<float>({2, 4}, cpu_place);
  float arr3[8] = {0, 1, 2, 3, 4, 5, 6, 7};
  memcpy(input3_ptr, arr3, 8 * sizeof(float));

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input2, gpu_place, &input2_gpu);
  paddle::framework::TensorCopySync(input3, gpu_place, &input3_gpu);
  float* a = input1_gpu.data<float>();
  float* b = input2_gpu.data<float>();
  float* c = input3_gpu.mutable_data<float>(gpu_place);

  GetBlas<float>(context).GEMM(false, true, m, n, k, 1, a, 3, b + 3, 3, 1,
                               c + 1, 4);

  paddle::framework::TensorCopySync(input3_gpu, cpu_place, &input3);

  context.Wait();
  EXPECT_EQ(input3_ptr[0], 0);
  EXPECT_EQ(input3_ptr[1], 24);
  EXPECT_EQ(input3_ptr[2], 28);
  EXPECT_EQ(input3_ptr[3], 32);
  EXPECT_EQ(input3_ptr[4], 4);
  EXPECT_EQ(input3_ptr[5], 73);
  EXPECT_EQ(input3_ptr[6], 86);
  EXPECT_EQ(input3_ptr[7], 99);
}

TEST(math_function, gemm_trans_cublas_fp16) {
  paddle::framework::Tensor input1;
  paddle::framework::Tensor input2;
  paddle::framework::Tensor input3;
  paddle::framework::Tensor input1_gpu;
  paddle::framework::Tensor input2_gpu;
  paddle::framework::Tensor input3_gpu;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  // fp16 GEMM in cublas requires GPU compute capability >= 53
  if (context.GetComputeCapability() < 53) {
    return;
  }

  int m = 2;
  int n = 3;
  int k = 3;
  paddle::platform::float16* input1_ptr =
      input1.mutable_data<paddle::platform::float16>({2, 3}, cpu_place);
  fill_fp16_data(input1_ptr, input1.numel(), {0, 1, 2, 3, 4, 5});
  paddle::platform::float16* input2_ptr =
      input2.mutable_data<paddle::platform::float16>({4, 3}, cpu_place);
  fill_fp16_data(input2_ptr, input2.numel(),
                 {0, 4, 8, 1, 5, 9, 2, 6, 10, 3, 7, 11});
  paddle::platform::float16* input3_ptr =
      input3.mutable_data<paddle::platform::float16>({2, 4}, cpu_place);
  fill_fp16_data(input3_ptr, input3.numel(), {0, 1, 2, 3, 4, 5, 6, 7});

  paddle::framework::TensorCopySync(input1, gpu_place, &input1_gpu);
  paddle::framework::TensorCopySync(input2, gpu_place, &input2_gpu);
  paddle::framework::TensorCopySync(input3, gpu_place, &input3_gpu);
  paddle::platform::float16* a = input1_gpu.data<paddle::platform::float16>();
  paddle::platform::float16* b = input2_gpu.data<paddle::platform::float16>();
  paddle::platform::float16* c =
      input3_gpu.mutable_data<paddle::platform::float16>(gpu_place);

  GetBlas<paddle::platform::float16>(context).GEMM(
      false, true, m, n, k, static_cast<paddle::platform::float16>(1), a, 3,
      b + 3, 3, static_cast<paddle::platform::float16>(1), c + 1, 4);

  paddle::framework::TensorCopySync(input3_gpu, cpu_place, &input3);

  context.Wait();
  EXPECT_EQ(static_cast<float>(input3_ptr[0]), 0);
  EXPECT_EQ(static_cast<float>(input3_ptr[1]), 24);
  EXPECT_EQ(static_cast<float>(input3_ptr[2]), 28);
  EXPECT_EQ(static_cast<float>(input3_ptr[3]), 32);
  EXPECT_EQ(static_cast<float>(input3_ptr[4]), 4);
  EXPECT_EQ(static_cast<float>(input3_ptr[5]), 73);
  EXPECT_EQ(static_cast<float>(input3_ptr[6]), 86);
  EXPECT_EQ(static_cast<float>(input3_ptr[7]), 99);
}

template <typename T>
void GemvTest(int m, int n, bool trans) {
  paddle::framework::Tensor mat_a;
  paddle::framework::Tensor vec_b;
  paddle::framework::Tensor vec_c;

  paddle::platform::CPUPlace cpu_place;
  paddle::platform::CUDAPlace gpu_place(0);
  paddle::platform::CUDADeviceContext context(gpu_place);

  T* data_a = mat_a.mutable_data<T>({m, n}, cpu_place);
  T* data_b = vec_b.mutable_data<T>({trans ? m : n}, cpu_place);
  T* data_c = vec_c.mutable_data<T>({trans ? n : m}, cpu_place);

  paddle::framework::Tensor g_mat_a;
  paddle::framework::Tensor g_vec_b;
  paddle::framework::Tensor g_vec_c;
  T* g_data_a = g_mat_a.mutable_data<T>(mat_a.dims(), gpu_place);
  T* g_data_b = g_vec_b.mutable_data<T>(vec_b.dims(), gpu_place);
  T* g_data_c = g_vec_c.mutable_data<T>(vec_c.dims(), gpu_place);

  for (int i = 0; i < mat_a.numel(); ++i) {
    data_a[i] = static_cast<T>(i);
  }
  for (int i = 0; i < vec_b.numel(); ++i) {
    data_b[i] = static_cast<T>(i);
  }

  paddle::framework::TensorCopySync(mat_a, gpu_place, &g_mat_a);
  paddle::framework::TensorCopySync(vec_b, gpu_place, &g_vec_b);

  GetBlas<T>(context).GEMV(trans, static_cast<int>(m), static_cast<int>(n), 1.,
                           g_data_a, g_data_b, 0., g_data_c);

  paddle::framework::TensorCopySync(g_vec_c, cpu_place, &vec_c);

  if (!trans) {
    for (int i = 0; i < m; ++i) {
      T sum = 0.0;
      for (int j = 0; j < n; ++j) {
        sum += data_a[i * n + j] * data_b[j];
      }
      ASSERT_FLOAT_EQ(data_c[i], sum);
    }
  } else {
    for (int i = 0; i < n; ++i) {
      T sum = 0.0;
      for (int j = 0; j < m; ++j) {
        sum += data_a[j * n + i] * data_b[j];
      }
      ASSERT_FLOAT_EQ(data_c[i], sum);
    }
  }
}

TEST(math_function, gemv) {
  GemvTest<float>(3, 13, false);
  GemvTest<double>(3, 13, false);
  GemvTest<float>(3, 13, true);
  GemvTest<double>(3, 13, true);
}
