/* Copyright (c) 2018 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/concat_and_split.h"

#include "paddle/phi/kernels/gpu/concat_and_split.h"
namespace paddle {
namespace operators {
namespace math {

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */
template <typename T>
class ConcatFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const std::vector<framework::Tensor>& input, int axis,
                  framework::Tensor* output) {
    std::vector<pten::DenseTensor> pt_input{input.begin(), input.end()};

    pten::ConcatImpl<T, platform::CUDADeviceContext>(context, pt_input, axis,
                                                     output);
  }
};

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */
template <typename T>
class SplitFunctor<platform::CUDADeviceContext, T> {
 public:
  SplitFunctor();
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input,
                  const std::vector<const framework::Tensor*>& ref_inputs,
                  int axis, std::vector<framework::Tensor*>* outputs) {
    std::vector<const pten::DenseTensor*> pt_ref_inputs{ref_inputs.begin(),
                                                        ref_inputs.end()};
    std::vector<pten::DenseTensor*> pt_outputs{outputs->begin(),
                                               outputs->end()};
    pten::SplitImpl<T, platform::CUDADeviceContext>(
        context, input, pt_ref_inputs, axis, &pt_outputs);
  }
};

#define DEFINE_FUNCTOR(type)                                       \
  template class ConcatFunctor<platform::CUDADeviceContext, type>; \
  template class SplitFunctor<platform::CUDADeviceContext, type>

FOR_ALL_TYPES(DEFINE_FUNCTOR);

}  // namespace math
}  // namespace operators
}  // namespace paddle
