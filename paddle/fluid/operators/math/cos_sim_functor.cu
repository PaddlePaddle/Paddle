#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/cos_sim_functor.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__global__ void CosSimDyKernel(const T* x_norm, const T* y_norm, const T* x,
                               const T* y, const T* z, const T* dz,
                               const size_t rows, const size_t cols, T* dy) {
  int grid_size = blockDim.x * gridDim.x;
  T y_norm_data = y_norm[0];
  for (int row_id = blockIdx.x * blockDim.x + threadIdx.x; row_id < rows;
       row_id += grid_size) {
    T xy_norm_prod = x_norm[row_id] * y_norm_data;
    T dz_data = dz[row_id];
    T z_data = z[row_id];
    const T* x_data = x + cols * row_id;
    T reciprocal_xy_norm_prod = 1 / xy_norm_prod;

    T y_norm_square = y_norm_data * y_norm_data;
    T reciprocal_y_norm_square = 1 / y_norm_square;
    for (size_t i = 0; i < cols; ++i) {
      T dy_data = dz_data * (x_data[i] * reciprocal_xy_norm_prod -
                             z_data * y[i] * reciprocal_y_norm_square);
      platform::CudaAtomicAdd(dy + i, dy_data);
    }
  }
}

template <typename T>
struct CosSimDyFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx, const T* x_norm,
                  const T* y_norm, const T* x, const T* y, const T* z,
                  const T* dz, const size_t rows, const size_t cols,
                  T* dy) const {
    const int block_size = 512;
    dim3 threads(block_size, 1);
    dim3 grid((rows + block_size - 1) / block_size, 1);
    CosSimDyKernel<T><<<grid, threads, 0, ctx.stream()>>>(
        x_norm, y_norm, x, y, z, dz, rows, cols, dy);
  }
};

template struct CosSimDyFunctor<platform::CUDADeviceContext, float>;
template struct CosSimDyFunctor<platform::CUDADeviceContext, double>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
