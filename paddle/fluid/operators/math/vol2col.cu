#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "paddle/fluid/operators/math/vol2col.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <class T>
__global__ void vol2col(int num_kernels, const T* data_vol, int depth,
                        int height, int width, int dilation_d, int dilation_h,
                        int dilation_w, int filter_depth, int filter_height,
                        int filter_width, int stride_depth, int stride_height,
                        int stride_width, int padding_depth, int padding_height,
                        int padding_width, int output_detph, int output_height,
                        int output_width, T* data_col) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < num_kernels;
       index += blockDim.x * gridDim.x) {
    int w_out = index % output_width;
    int h_out = (index / output_width) % output_height;
    int d_out = (index / output_width / output_height) % output_detph;
    int channel_in = index / output_width / output_height / output_detph;
    int channel_out = channel_in * filter_depth * filter_height * filter_width;
    int w_in = w_out * stride_width - padding_width;
    int h_in = h_out * stride_height - padding_height;
    int d_in = d_out * stride_depth - padding_depth;

    data_col += ((channel_out * output_detph + d_out) * output_height + h_out) *
                    output_width +
                w_out;
    data_vol += ((channel_in * depth + d_in) * height + h_in) * width + w_in;
    for (int k = 0; k < filter_depth; ++k) {
      for (int i = 0; i < filter_height; ++i) {
        for (int j = 0; j < filter_width; ++j) {
          int d = d_in + k * dilation_d;
          int h = h_in + i * dilation_h;
          int w = w_in + j * dilation_w;
          int col_idx = (k * dilation_d * height + i * dilation_h) * width +
                        j * dilation_w;
          *data_col = (d >= 0 && d < depth && h >= 0 && h < height && w >= 0 &&
                       w < width)
                          ? data_vol[col_idx]
                          : 0;
          data_col += output_detph * output_height * output_width;
        }
      }
    }
  }
}

/*
 * im = [input_channels,intpu_depth, input_height, input_width]
 * col =
 *   [input_channels, filter_depth, filter_height, filter_width,
 *                    output_depth, output_height, output_width]
 */
template <class T>
class Vol2ColFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& vol,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* col) const {
    PADDLE_ENFORCE_EQ(vol.dims().size(), 4);
    PADDLE_ENFORCE_EQ(col->dims().size(), 7);

    int input_channels = vol.dims()[0];
    int input_depth = vol.dims()[1];
    int input_height = vol.dims()[2];
    int input_width = vol.dims()[3];
    int filter_depth = col->dims()[1];
    int filter_height = col->dims()[2];
    int filter_width = col->dims()[3];
    int output_depth = col->dims()[4];
    int output_height = col->dims()[5];
    int output_width = col->dims()[6];

    PADDLE_ENFORCE_EQ((input_depth + 2 * paddings[0] -
                       ((dilations[0] * (filter_depth - 1) + 1))) /
                              strides[0] +
                          1,
                      output_depth,
                      "input_depth and output_depth are "
                      "Mismatching.");
    PADDLE_ENFORCE_EQ((input_height + 2 * paddings[1] -
                       ((dilations[1] * (filter_height - 1) + 1))) /
                              strides[1] +
                          1,
                      output_height,
                      "input_height and output_height are "
                      "Mismatching.");
    PADDLE_ENFORCE_EQ((input_width + 2 * paddings[2] -
                       ((dilations[2] * (filter_width - 1) + 1))) /
                              strides[2] +
                          1,
                      output_width,
                      "input_width and output_width are "
                      "Mismatching.");

    int num_outputs =
        input_channels * output_depth * output_height * output_width;

    const int threads = 1024;
    const int blocks = (num_outputs + 1024 - 1) / 1024;
    vol2col<T><<<blocks, threads, 0, context.stream()>>>(
        num_outputs, vol.data<T>(), input_depth, input_height, input_width,
        dilations[0], dilations[1], dilations[2], filter_depth, filter_height,
        filter_width, strides[0], strides[1], strides[2], paddings[0],
        paddings[1], paddings[2], output_depth, output_height, output_width,
        col->data<T>());
  }
};

template <class T>
__global__ void col2vol(int num_kernels, const T* data_col, int depth,
                        int height, int width, int dilation_d, int dilation_h,
                        int dilation_w, int filter_depth, int filter_height,
                        int filter_width, int stride_depth, int stride_height,
                        int stride_width, int padding_depth, int padding_height,
                        int padding_width, int output_detph, int output_height,
                        int output_width, T* data_vol) {
  const int d_filter_depth = dilation_d * (filter_depth - 1) + 1;
  const int d_filter_height = dilation_h * (filter_height - 1) + 1;
  const int d_filter_width = dilation_w * (filter_width - 1) + 1;

  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < num_kernels;
       index += blockDim.x * gridDim.x) {
    T src_val = 0;
    int w = index % width + padding_width;
    int h = (index / width) % height + padding_height;
    int d = (index / width / height) % depth + padding_depth;
    int c = index / width / height / depth;

    // compute the start and end of the output
    int w_col_start =
        (w < d_filter_width) ? 0 : (w - d_filter_width) / stride_width + 1;
    int w_col_end = min(w / stride_width + 1, output_width);
    int h_col_start =
        (h < d_filter_height) ? 0 : (h - d_filter_height) / stride_height + 1;
    int h_col_end = min(h / stride_height + 1, output_height);
    int d_col_start =
        (d < d_filter_depth) ? 0 : (d - d_filter_depth) / stride_depth + 1;
    int d_col_end = min(d / stride_depth + 1, output_detph);

    for (int d_col = d_col_start; d_col < d_col_end; ++d_col) {
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          int d_off = (d - d_col * stride_depth);
          int h_off = (h - h_col * stride_height);
          int w_off = (w - w_col * stride_width);
          if (d_off % dilation_d == 0 && h_off % dilation_h == 0 &&
              w_off % dilation_w == 0) {
            d_off /= dilation_d;
            h_off /= dilation_h;
            w_off /= dilation_w;

            int data_col_index =
                (((((c * filter_depth + d_off) * filter_height + h_off) *
                       filter_width +
                   w_off)));
            data_col_index =
                ((data_col_index * output_detph + d_col) * output_height +
                 h_col) *
                    output_width +
                w_col;
            src_val += data_col[data_col_index];
          }
        }
      }
    }
    data_vol[index] = src_val;
  }
}

/*
 * im = [input_channels, input_depth, input_height, input_width]
 * col =
 *   [input_channels, filter_depth, filter_height, filter_width,
 *                    output_depth, output_height, output_width]
 */
template <class T>
class Col2VolFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& col,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings,
                  framework::Tensor* vol) const {
    PADDLE_ENFORCE_EQ(vol->dims().size(), 4);
    PADDLE_ENFORCE_EQ(col.dims().size(), 7);

    int input_channels = vol->dims()[0];
    int input_depth = vol->dims()[1];
    int input_height = vol->dims()[2];
    int input_width = vol->dims()[3];
    int filter_depth = col.dims()[1];
    int filter_height = col.dims()[2];
    int filter_width = col.dims()[3];
    int output_depth = col.dims()[4];
    int output_height = col.dims()[5];
    int output_width = col.dims()[6];

    PADDLE_ENFORCE_EQ((input_depth + 2 * paddings[0] -
                       ((dilations[0] * (filter_depth - 1) + 1))) /
                              strides[0] +
                          1,
                      output_depth,
                      "input_depth and output_depth are "
                      "Mismatching.");
    PADDLE_ENFORCE_EQ((input_height + 2 * paddings[1] -
                       ((dilations[1] * (filter_height - 1) + 1))) /
                              strides[1] +
                          1,
                      output_height,
                      "input_height and output_height are "
                      "Mismatching.");
    PADDLE_ENFORCE_EQ((input_width + 2 * paddings[2] -
                       ((dilations[2] * (filter_width - 1) + 1))) /
                              strides[2] +
                          1,
                      output_width,
                      "input_width and output_width are "
                      "Mismatching.");

    int num_kernels = input_channels * input_depth * input_height * input_width;

    const int threads = 1024;
    const int blocks = (num_kernels + 1024 - 1) / 1024;

    col2vol<T><<<blocks, threads, 0, context.stream()>>>(
        num_kernels, col.data<T>(), input_depth, input_height, input_width,
        dilations[0], dilations[1], dilations[2], filter_depth, filter_height,
        filter_width, strides[0], strides[1], strides[2], paddings[0],
        paddings[1], paddings[2], output_depth, output_height, output_width,
        vol->data<T>());
  }
};

template class Vol2ColFunctor<platform::CUDADeviceContext, float>;
template class Vol2ColFunctor<platform::CUDADeviceContext, double>;
template class Col2VolFunctor<platform::CUDADeviceContext, float>;
template class Col2VolFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
