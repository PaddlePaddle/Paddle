#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "paddle/fluid/operators/math/vol2col.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <class T>
__global__ void vol2col(int num_kernels, const T* data_vol, int depth,
                        int height, int width, int dilation_d, int dilation_h,
                        int dilation_w, int filter_depth, int filter_height,
                        int filter_width, int stride_depth, int stride_height,
                        int stride_width, int padding_depth, int padding_height,
                        int padding_width, int output_detph, int output_height,
                        int output_width, T* data_col,
                        const DataLayout data_layout) {
  int input_channels =
      num_kernels / output_detph / output_height / output_width;
  int channels_col =
      input_channels * filter_depth * filter_height * filter_width;
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < num_kernels;
       index += blockDim.x * gridDim.x) {
    int w_out = index % output_width;
    int h_out = (index / output_width) % output_height;
    int d_out = (index / output_width / output_height) % output_detph;
    int channel_in = index / output_width / output_height / output_detph;
    int channel_out = channel_in * filter_depth * filter_height * filter_width;
    int w_in = w_out * stride_width - padding_width;
    int h_in = h_out * stride_height - padding_height;
    int d_in = d_out * stride_depth - padding_depth;

    data_col += ((channel_out * output_detph + d_out) * output_height + h_out) *
                    output_width +
                w_out;
    for (int k = 0; k < filter_depth; ++k) {
      for (int i = 0; i < filter_height; ++i) {
        for (int j = 0; j < filter_width; ++j) {
          int d = d_in + k * dilation_d;
          int h = h_in + i * dilation_h;
          int w = w_in + j * dilation_w;
          int vol_idx;
          if (data_layout != DataLayout::kNHWC) {
            vol_idx = ((channel_in * depth + d) * height + h) * width + w;
          } else {
            vol_idx =
                ((d * height + h) * width + w) * input_channels + channel_in;
          }
          *data_col = (d >= 0 && d < depth && h >= 0 && h < height && w >= 0 &&
                       w < width)
                          ? data_vol[vol_idx]
                          : 0;
          data_col += output_detph * output_height * output_width;
        }
      }
    }
  }
}

/*
 * im = [input_channels,intpu_depth, input_height, input_width] for
 * channels_first
 * im = [input_depth, input_height, input_width, input_channels] for
 * channels_last
 * col =
 *   [input_channels, filter_depth, filter_height, filter_width,
 *                    output_depth, output_height, output_width]
 */
template <class T>
class Vol2ColFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& vol,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, framework::Tensor* col,
                  const DataLayout data_layout) const {
    PADDLE_ENFORCE_EQ(vol.dims().size(), 4,
                      platform::errors::InvalidArgument(
                          "The dimension of  vol should be 4, but received %d.",
                          vol.dims().size()));
    PADDLE_ENFORCE_EQ(col->dims().size(), 7,
                      platform::errors::InvalidArgument(
                          "The dimension of col should be 7, but received %d.",
                          col->dims().size()));

    int input_channels =
        (data_layout != DataLayout::kNHWC ? vol.dims()[0] : vol.dims()[3]);
    int input_depth =
        (data_layout != DataLayout::kNHWC ? vol.dims()[1] : vol.dims()[0]);
    int input_height =
        (data_layout != DataLayout::kNHWC ? vol.dims()[2] : vol.dims()[1]);
    int input_width =
        (data_layout != DataLayout::kNHWC ? vol.dims()[3] : vol.dims()[2]);
    int filter_depth = col->dims()[1];
    int filter_height = col->dims()[2];
    int filter_width = col->dims()[3];
    int output_depth = col->dims()[4];
    int output_height = col->dims()[5];
    int output_width = col->dims()[6];

    bool paddings_size_is_6 = (paddings.size() == 6);
    int pad_d_forth = paddings_size_is_6 ? paddings[0] : paddings[0];
    int pad_d_back = paddings_size_is_6 ? paddings[1] : paddings[0];
    int pad_h_up = paddings_size_is_6 ? paddings[2] : paddings[1];
    int pad_h_down = paddings_size_is_6 ? paddings[3] : paddings[1];
    int pad_w_left = paddings_size_is_6 ? paddings[4] : paddings[2];
    int pad_w_right = paddings_size_is_6 ? paddings[5] : paddings[2];
    auto input_depth_tmp = (input_depth + pad_d_forth + pad_d_back -
                            ((dilations[0] * (filter_depth - 1) + 1))) /
                               strides[0] +
                           1;
    PADDLE_ENFORCE_EQ(
        input_depth_tmp, output_depth,
        platform::errors::InvalidArgument(
            "input_depth(%d) and output_depth(%d) are mismatching.",
            input_depth_tmp, output_depth));
    auto input_height_tmp = (input_height + pad_h_up + pad_h_down -
                             ((dilations[1] * (filter_height - 1) + 1))) /
                                strides[1] +
                            1;
    PADDLE_ENFORCE_EQ(
        input_height_tmp, output_height,
        platform::errors::InvalidArgument(
            "input_height(%d) and output_height(%d) are mismatching.",
            input_height_tmp, output_height));
    auto input_width_tmp = (input_width + pad_w_left + pad_w_right -
                            ((dilations[2] * (filter_width - 1) + 1))) /
                               strides[2] +
                           1;
    PADDLE_ENFORCE_EQ(
        input_width_tmp, output_width,
        platform::errors::InvalidArgument(
            "input_width(%d) and output_width(%d) are mismatching.",
            input_width_tmp, output_width));

    int num_outputs =
        input_channels * output_depth * output_height * output_width;

    const int threads = 1024;
    const int blocks = (num_outputs + 1024 - 1) / 1024;
    vol2col<T><<<blocks, threads, 0, context.stream()>>>(
        num_outputs, vol.data<T>(), input_depth, input_height, input_width,
        dilations[0], dilations[1], dilations[2], filter_depth, filter_height,
        filter_width, strides[0], strides[1], strides[2], pad_d_forth, pad_h_up,
        pad_w_left, output_depth, output_height, output_width, col->data<T>(),
        data_layout);
  }
};

template <class T>
__global__ void col2vol(int num_kernels, const T* data_col, int depth,
                        int height, int width, int dilation_d, int dilation_h,
                        int dilation_w, int filter_depth, int filter_height,
                        int filter_width, int stride_depth, int stride_height,
                        int stride_width, int padding_depth, int padding_height,
                        int padding_width, int output_detph, int output_height,
                        int output_width, T* data_vol,
                        const DataLayout data_layout) {
  const int d_filter_depth = dilation_d * (filter_depth - 1) + 1;
  const int d_filter_height = dilation_h * (filter_height - 1) + 1;
  const int d_filter_width = dilation_w * (filter_width - 1) + 1;

  int input_channels = num_kernels / depth / height / width;
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < num_kernels;
       index += blockDim.x * gridDim.x) {
    T src_val = 0;
    int w = (data_layout != DataLayout::kNHWC
                 ? index % width + padding_width
                 : (index / input_channels) % width + padding_width);
    int h = (data_layout != DataLayout::kNHWC
                 ? (index / width) % height + padding_height
                 : (index / input_channels / width) % height + padding_height);
    int d = (data_layout != DataLayout::kNHWC
                 ? (index / width / height) % depth + padding_depth
                 : index / input_channels / width / height + padding_depth);
    int c = (data_layout != DataLayout::kNHWC ? index / width / height / depth
                                              : index % input_channels);

    // compute the start and end of the output
    int w_col_start =
        (w < d_filter_width) ? 0 : (w - d_filter_width) / stride_width + 1;
    int w_col_end = min(w / stride_width + 1, output_width);
    int h_col_start =
        (h < d_filter_height) ? 0 : (h - d_filter_height) / stride_height + 1;
    int h_col_end = min(h / stride_height + 1, output_height);
    int d_col_start =
        (d < d_filter_depth) ? 0 : (d - d_filter_depth) / stride_depth + 1;
    int d_col_end = min(d / stride_depth + 1, output_detph);

    for (int d_col = d_col_start; d_col < d_col_end; ++d_col) {
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          int d_off = (d - d_col * stride_depth);
          int h_off = (h - h_col * stride_height);
          int w_off = (w - w_col * stride_width);
          if (d_off % dilation_d == 0 && h_off % dilation_h == 0 &&
              w_off % dilation_w == 0) {
            d_off /= dilation_d;
            h_off /= dilation_h;
            w_off /= dilation_w;

            int data_col_index =
                (((((c * filter_depth + d_off) * filter_height + h_off) *
                       filter_width +
                   w_off)));
            data_col_index =
                ((data_col_index * output_detph + d_col) * output_height +
                 h_col) *
                    output_width +
                w_col;
            src_val += data_col[data_col_index];
          }
        }
      }
    }
    data_vol[index] = src_val;
  }
}

/*
 * im = [input_channels,intpu_depth, input_height, input_width] for
 * channels_first
 * im = [input_depth, input_height, input_width, input_channels] for
 * channels_last
 * col =
 *   [input_channels, filter_depth, filter_height, filter_width,
 *                    output_depth, output_height, output_width]
 */
template <class T>
class Col2VolFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& col,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, framework::Tensor* vol,
                  const DataLayout data_layout) const {
    PADDLE_ENFORCE_EQ(vol->dims().size(), 4,
                      platform::errors::InvalidArgument(
                          "The dimension of vol  should be 4, but received %d.",
                          vol->dims().size()));
    PADDLE_ENFORCE_EQ(col.dims().size(), 7,
                      platform::errors::InvalidArgument(
                          "The dimension of col  should be 7, but received %d.",
                          col.dims().size()));

    int input_channels =
        (data_layout != DataLayout::kNHWC ? vol->dims()[0] : vol->dims()[3]);
    int input_depth =
        (data_layout != DataLayout::kNHWC ? vol->dims()[1] : vol->dims()[0]);
    int input_height =
        (data_layout != DataLayout::kNHWC ? vol->dims()[2] : vol->dims()[1]);
    int input_width =
        (data_layout != DataLayout::kNHWC ? vol->dims()[3] : vol->dims()[2]);
    int filter_depth = col.dims()[1];
    int filter_height = col.dims()[2];
    int filter_width = col.dims()[3];
    int output_depth = col.dims()[4];
    int output_height = col.dims()[5];
    int output_width = col.dims()[6];

    bool paddings_size_is_6 = (paddings.size() == 6);
    int pad_d_forth = paddings_size_is_6 ? paddings[0] : paddings[0];
    int pad_d_back = paddings_size_is_6 ? paddings[1] : paddings[0];
    int pad_h_up = paddings_size_is_6 ? paddings[2] : paddings[1];
    int pad_h_down = paddings_size_is_6 ? paddings[3] : paddings[1];
    int pad_w_left = paddings_size_is_6 ? paddings[4] : paddings[2];
    int pad_w_right = paddings_size_is_6 ? paddings[5] : paddings[2];

    auto input_depth_tmp = (input_depth + pad_d_forth + pad_d_back -
                            ((dilations[0] * (filter_depth - 1) + 1))) /
                               strides[0] +
                           1;
    PADDLE_ENFORCE_EQ(
        input_depth_tmp, output_depth,
        platform::errors::InvalidArgument(
            "input_depth(%d) and output_depth(%d) are mismatching.",
            input_depth_tmp, output_depth));
    auto input_height_tmp = (input_height + pad_h_up + pad_h_down -
                             ((dilations[1] * (filter_height - 1) + 1))) /
                                strides[1] +
                            1;
    PADDLE_ENFORCE_EQ(
        input_height_tmp, output_height,
        platform::errors::InvalidArgument(
            "input_height(%d) and output_height(%d) are mismatching.",
            input_height_tmp, output_height));
    auto input_width_tmp = (input_width + pad_w_left + pad_w_right -
                            ((dilations[2] * (filter_width - 1) + 1))) /
                               strides[2] +
                           1;
    PADDLE_ENFORCE_EQ(
        input_width_tmp, output_width,
        platform::errors::InvalidArgument(
            "input_width(%d) and output_width(%d) are mismatching.",
            input_width_tmp, output_width));

    int num_kernels = input_channels * input_depth * input_height * input_width;

    const int threads = 1024;
    const int blocks = (num_kernels + 1024 - 1) / 1024;

    col2vol<T><<<blocks, threads, 0, context.stream()>>>(
        num_kernels, col.data<T>(), input_depth, input_height, input_width,
        dilations[0], dilations[1], dilations[2], filter_depth, filter_height,
        filter_width, strides[0], strides[1], strides[2], pad_d_forth, pad_h_up,
        pad_w_left, output_depth, output_height, output_width, vol->data<T>(),
        data_layout);
  }
};

template class Vol2ColFunctor<platform::CUDADeviceContext, float>;
template class Vol2ColFunctor<platform::CUDADeviceContext, double>;
template class Col2VolFunctor<platform::CUDADeviceContext, float>;
template class Col2VolFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
