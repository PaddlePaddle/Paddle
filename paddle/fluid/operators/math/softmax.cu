/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU

#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/softmax.h"
#include "paddle/fluid/operators/math/softmax_impl.h"
#include "paddle/fluid/platform/cudnn_helper.h"

namespace paddle {
namespace operators {
namespace math {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;

template <typename T>
void SoftmaxCUDNNFunctor<T>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor* X,
    framework::Tensor* Y) {
  // ------------------- cudnn descriptors ---------------------
  ScopedTensorDescriptor xDesc;
  ScopedTensorDescriptor yDesc;
  std::vector<int> cudnn_tensor_dims = framework::vectorize2int(X->dims());
  DataLayout layout = DataLayout::kNCHW;
  if (cudnn_tensor_dims.size() == 5) {
    layout = DataLayout::kNCDHW;
  }
  // NOTE(*) : cudnn softmax only support >= 4D Tensor,
  // fill 1 at unused dims
  if (cudnn_tensor_dims.size() <= 2) {
    cudnn_tensor_dims.resize(4, 1);
  }
  hipdnnTensorDescriptor_t cudnn_x_desc =
      xDesc.descriptor<T>(layout, cudnn_tensor_dims);
  hipdnnTensorDescriptor_t cudnn_y_desc =
      xDesc.descriptor<T>(layout, cudnn_tensor_dims);
  PADDLE_ENFORCE(platform::dynload::hipdnnSoftmaxForward(
      context.cudnn_handle(), HIPDNN_SOFTMAX_ACCURATE,
      HIPDNN_SOFTMAX_MODE_INSTANCE, CudnnDataType<T>::kOne(), cudnn_x_desc,
      X->data<T>(), CudnnDataType<T>::kZero(), cudnn_y_desc,
      Y->mutable_data<T>(context.GetPlace())));
}

template <typename T>
void SoftmaxGradCUDNNFunctor<T>::operator()(
    const platform::CUDADeviceContext& context, const framework::Tensor* Y,
    const framework::Tensor* YGrad, framework::Tensor* XGrad) {
  // ------------------- cudnn descriptors ---------------------
  ScopedTensorDescriptor yDesc;
  ScopedTensorDescriptor dyDesc;
  ScopedTensorDescriptor dxDesc;
  std::vector<int> cudnn_tensor_dims = framework::vectorize2int(Y->dims());
  DataLayout layout = DataLayout::kNCHW;
  if (cudnn_tensor_dims.size() == 5) {
    layout = DataLayout::kNCDHW;
  }
  // NOTE(*) : cudnn softmax only support >= 4D Tensor,
  // fill 1 at unused dims
  if (cudnn_tensor_dims.size() <= 2) {
    cudnn_tensor_dims.resize(4, 1);
  }
  hipdnnTensorDescriptor_t cudnn_y_desc =
      yDesc.descriptor<T>(layout, cudnn_tensor_dims);
  hipdnnTensorDescriptor_t cudnn_xgrad_desc =
      dxDesc.descriptor<T>(layout, cudnn_tensor_dims);
  hipdnnTensorDescriptor_t cudnn_ygrad_desc =
      dyDesc.descriptor<T>(layout, cudnn_tensor_dims);
  PADDLE_ENFORCE(platform::dynload::hipdnnSoftmaxBackward(
      context.cudnn_handle(), HIPDNN_SOFTMAX_ACCURATE,
      HIPDNN_SOFTMAX_MODE_INSTANCE, CudnnDataType<T>::kOne(), cudnn_y_desc,
      Y->data<T>(), cudnn_ygrad_desc, YGrad->data<T>(),
      CudnnDataType<T>::kZero(), cudnn_xgrad_desc,
      XGrad->mutable_data<T>(context.GetPlace())));
}

template class SoftmaxCUDNNFunctor<float>;
template class SoftmaxCUDNNFunctor<double>;
template class SoftmaxGradCUDNNFunctor<float>;
template class SoftmaxGradCUDNNFunctor<double>;

template class SoftmaxFunctor<platform::CUDADeviceContext, float>;
template class SoftmaxFunctor<platform::CUDADeviceContext, double>;
template class SoftmaxGradFunctor<platform::CUDADeviceContext, float>;
template class SoftmaxGradFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
