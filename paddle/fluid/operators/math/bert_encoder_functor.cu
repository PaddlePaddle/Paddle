#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hip/hip_runtime.h>
#include <algorithm>
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/platform/enforce.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T, int TPB>
__device__ inline void LayerNormSmall(T val, const kvp<T> &thread_data,
                                      const int ld, const int idx,
                                      const float *bias, const float *scale,
                                      T *output, T eps) {
  using BlockReduce = hipcub::BlockReduce<kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  if (threadIdx.x < ld) {
    const T g(scale[threadIdx.x]);
    const T b(bias[threadIdx.x]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, int TPB>
__device__ inline void LayerNorm(const kvp<T> &thread_data, const int ld,
                                 const int offset, const float *bias,
                                 const float *scale, T *output, T eps) {
  using BlockReduce = hipcub::BlockReduce<kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    const T val = output[idx];
    const T g(scale[i]);
    const T b(bias[i]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, typename T2, int TPB>
__device__ inline void LayerNorm2(const kvp<T> &thread_data, const int ld,
                                  const int offset, const float2 *bias,
                                  const float2 *scale, T2 *output, T eps) {
  using BlockReduce = hipcub::BlockReduce<kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    T2 val = output[idx];
    const float2 g = scale[i];
    const float2 b = bias[i];
    val.x = T(g.x) * (val.x - mu) * rsigma + T(b.x);
    val.y = T(g.y) * (val.y - mu) * rsigma + T(b.y);
    output[idx] = val;
  }
}

template <typename T, unsigned TPB>
__global__ void EmbEltwiseLayernormKernel(int hidden, const int64_t *ids,
                                          const float *scale, const float *bias,
                                          const int64_t *embs, T *output,
                                          float eps, int input_num) {
  hipcub::Sum pair_sum;
  // blockIdx.x: position in the sequence
  // blockIdx.y: batch
  // gridDim.x: Seq
  // gridDim.y: Batch

  extern __shared__ int64_t array_id[];

  const T rhidden = T(1.f) / T(hidden);
  const int64_t seq_pos = blockIdx.y + blockIdx.x * gridDim.y;
  if (threadIdx.x == 0) {
    for (int i = 0; i < input_num; ++i) {
      const int64_t *ids_p = reinterpret_cast<const int64_t *>(ids[i]);
      array_id[i] = ids_p[seq_pos];
    }
  }
  __syncthreads();

  const int64_t out_offset = seq_pos * hidden;

  kvp<T> thread_data(0, 0);

#pragma unroll
  for (int it = threadIdx.x; it < hidden; it += TPB) {
    T val = 0;
    for (int i = 0; i < input_num; ++i) {
      val += reinterpret_cast<const T *>(embs[i])[array_id[i] * hidden + it];
    }

    output[out_offset + it] = val;
    const T rhiddenval = rhidden * val;
    thread_data = pair_sum(thread_data, kvp<T>(rhiddenval, rhiddenval * val));
  }
  LayerNorm<T, TPB>(thread_data, hidden, out_offset, bias, scale, output, eps);
}

template <typename T>
void EmbEltwiseLayerNormFunctor<T>::operator()(
    int batch, int seq_len, int hidden, const int64_t *ids, const float *scale,
    const float *bias, const int64_t *embs, T *output, float eps, int input_num,
    hipStream_t stream) {
  const unsigned tpb = 256;
  const dim3 grid(seq_len, batch, 1);
  const dim3 block(tpb, 1, 1);
  int shared_bytes = input_num * sizeof(int64_t);
  EmbEltwiseLayernormKernel<T, tpb><<<grid, block, shared_bytes, stream>>>(
      hidden, ids, scale, bias, embs, output, eps, input_num);
}

template class EmbEltwiseLayerNormFunctor<float>;

#ifdef SUPPORTS_CUDA_FP16
template class EmbEltwiseLayerNormFunctor<half>;
#endif

template <typename T>
__global__ void SoftmaxKernelWithEltadd(T *qk_buf_, const T *bias_qk_,
                                        const int batch_size,
                                        const int head_num, const int seq_len,
                                        const unsigned mask) {
  int qk_offset = blockIdx.x * seq_len;
  assert(blockDim.x % 32 == 0);

  float tmp = threadIdx.x < seq_len
                  ? static_cast<float>(qk_buf_[threadIdx.x + qk_offset] +
                                       bias_qk_[threadIdx.x + qk_offset])
                  : -1e20f;
  float max_val = blockReduceMax<float>(tmp, mask);

  float qk_tmp = threadIdx.x < seq_len ? __expf(tmp - max_val) : 0.0f;
  float sum_val = blockReduceSum<float>(qk_tmp, mask);

  if (threadIdx.x < seq_len)
    qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / sum_val);
}

template <typename T>
__global__ void SoftmaxKernelWithEltadd2(T *qk_buf_, const T *bias_qk_,
                                         const int batch_size,
                                         const int head_num, const int seq_len,
                                         const unsigned mask) {
  int qk_offset = blockIdx.x * seq_len;
  int idx = threadIdx.x;
  assert(blockDim.x % 32 == 0);

  float2 tmp =
      idx < seq_len
          ? ToFloat2<T>(qk_buf_[idx + qk_offset] + bias_qk_[idx + qk_offset])
          : make_float2(-1e20f, -1e20f);
  float max_val = blockReduceMax<float>(max(tmp.x, tmp.y), mask);
  float2 qk_tmp = idx < seq_len ? make_float2(__expf(tmp.x - max_val),
                                              __expf(tmp.y - max_val))
                                : make_float2(0.f, 0.f);
  float sum_val = blockReduceSum<float>(qk_tmp.x + qk_tmp.y, mask) + 1e-6f;

  if (idx < seq_len) {
    qk_buf_[idx + qk_offset] =
        FloatsToPair<T>(qk_tmp.x / sum_val, qk_tmp.y / sum_val);
  }
}

template <typename T>
inline void MatMulWithHeadQK(const platform::CUDADeviceContext &context,
                             int head_num, int seq_len, int size_per_head,
                             int batch_size, bool q_trans, bool k_trans,
                             T *q_buf_, T *k_buf_, T *qk_buf_, const T *bias_qk,
                             T alpha, T beta) {
  CBLAS_TRANSPOSE transA = !q_trans ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = !k_trans ? CblasNoTrans : CblasTrans;

  typedef typename CUDATypeTraits<T>::TYPE run_type;
  auto blas =
      operators::math::GetBlas<platform::CUDADeviceContext, run_type>(context);
  auto stream = context.stream();

  blas.BatchedGEMM(
      transA, transB, seq_len, seq_len, size_per_head,
      static_cast<run_type>(alpha), reinterpret_cast<run_type *>(q_buf_),
      reinterpret_cast<run_type *>(k_buf_), static_cast<run_type>(beta),
      reinterpret_cast<run_type *>(qk_buf_), batch_size * head_num,
      seq_len * size_per_head, seq_len * size_per_head);

  int grid = batch_size * head_num * seq_len;
  int block = seq_len;

  // Align block to 32, also limit seq_len to max block size.
  PADDLE_ENFORCE_LE(seq_len, 1024, platform::errors::InvalidArgument(
                                       "seq_len should <= 1024, "
                                       "but received seq_len is:%d",
                                       seq_len));
  if (seq_len % 2 == 0) {
    block = (seq_len <= 64) ? 32 : ((seq_len + 63) / 64) * 32;
#ifdef SUPPORTS_CUDA_FP16
    if (std::is_same<T, float>::value) {
#endif
      SoftmaxKernelWithEltadd2<float2><<<grid, block, 0, stream>>>(
          reinterpret_cast<float2 *>(qk_buf_),
          reinterpret_cast<const float2 *>(bias_qk), batch_size, head_num,
          seq_len / 2, FINAL_MASK);
#ifdef SUPPORTS_CUDA_FP16
    } else {
      SoftmaxKernelWithEltadd2<__half2><<<grid, block, 0, stream>>>(
          reinterpret_cast<__half2 *>(qk_buf_),
          reinterpret_cast<const __half2 *>(bias_qk), batch_size, head_num,
          seq_len / 2, FINAL_MASK);
    }
#endif
  } else {
    block = (seq_len <= 32) ? 32 : ((seq_len + 31) / 32) * 32;
    SoftmaxKernelWithEltadd<T><<<grid, block, 0, stream>>>(
        qk_buf_, bias_qk, batch_size, head_num, seq_len, FINAL_MASK);
  }
}

template <typename T>
inline void MatMulWithHeadQKV(const platform::CUDADeviceContext &context,
                              int head_num, int seq_len, int size_per_head,
                              int batch_size, bool qk_trans, bool v_trans,
                              T *v_buf_, const T *qk_buf_, T *dst, T alpha,
                              T beta) {
  int m = batch_size * seq_len;
  int k = head_num * size_per_head;

  typedef typename CUDATypeTraits<T>::TYPE run_type;
  auto blas =
      operators::math::GetBlas<platform::CUDADeviceContext, run_type>(context);
  auto stream = context.stream();
  CBLAS_TRANSPOSE transA = !qk_trans ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = !v_trans ? CblasNoTrans : CblasTrans;

  blas.BatchedGEMM(
      transA, transB, seq_len, size_per_head, seq_len,
      static_cast<run_type>(alpha), reinterpret_cast<const run_type *>(qk_buf_),
      reinterpret_cast<run_type *>(v_buf_), static_cast<run_type>(beta),
      reinterpret_cast<run_type *>(dst), batch_size * head_num,
      seq_len * seq_len, seq_len * size_per_head);
}

template <typename T>
void MultiHeadGPUComputeFunctor<T>::operator()(
    const platform::CUDADeviceContext &dev_ctx, int batch, int seq_len,
    int head_num, int head_size, T *qkptr, const T *bias_qk_ptr, T *tptr,
    T alpha, T beta) {
  auto stream = dev_ctx.stream();
  const int tsize = batch * head_num * seq_len * head_size;

  T *qptr = tptr;
  T *kptr = qptr + tsize;
  T *vptr = kptr + tsize;
  // batch gemm stride, softmaxwithscale.
  MatMulWithHeadQK<T>(dev_ctx, head_num, seq_len, head_size, batch, false, true,
                      qptr, kptr, qkptr, bias_qk_ptr, alpha, beta);
  // batch gemm stride, transpose.
  MatMulWithHeadQKV<T>(dev_ctx, head_num, seq_len, head_size, batch, false,
                       false, vptr, qkptr, tptr, T(1.0), beta);
}

template class MultiHeadGPUComputeFunctor<float>;

#ifdef SUPPORTS_CUDA_FP16
template class MultiHeadGPUComputeFunctor<half>;
#endif

template <typename T, unsigned TPB>
__global__ void SkipLayerNormSmallKernel(int num, int hidden, const T *input1,
                                         const T *input2, T *output,
                                         const float *scale, const float *bias,
                                         float eps) {
  const T rld = T(1) / T(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  kvp<T> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  T val = 0;
  if (threadIdx.x < hidden) {
    val = input1[idx] + input2[idx];
    const T rldval = rld * val;
    thread_data = pair_sum(thread_data, kvp<T>(rldval, rldval * val));
  }
  LayerNormSmall<T, TPB>(val, thread_data, hidden, idx, bias, scale, output,
                         eps);
}

template <typename T, unsigned TPB>
__global__ void SkipLayerNormKernel(int num, int hidden, const T *input1,
                                    const T *input2, T *output,
                                    const float *scale, const float *bias,
                                    float eps) {
  const T rld = T(1) / T(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  kvp<T> thread_data(0, 0);

  for (int it = threadIdx.x; it < hidden; it += TPB) {
    const int idx = offset + it;
    const T val = input1[idx] + input2[idx];
    const T rldval = rld * val;
    thread_data = pair_sum(thread_data, kvp<T>(rldval, rldval * val));
    output[idx] = val;
  }
  LayerNorm<T, TPB>(thread_data, hidden, offset, bias, scale, output, eps);
}

template <typename T, typename T2, unsigned TPB>
__global__ void SkipLayerNormKernel2(int num, int hidden, const T2 *input1,
                                     const T2 *input2, T2 *output,
                                     const float2 *scale, const float2 *bias,
                                     float eps) {
  const T rld = T(0.5f / hidden);  // because hidden is hidden/2
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  kvp<T> thread_data(0, 0);

  for (int it = threadIdx.x; it < hidden; it += TPB) {
    const int idx = offset + it;
    const T2 val2 = input1[idx] + input2[idx];
    thread_data = pair_sum(
        thread_data, kvp<T>(rld * (val2.x + val2.y),
                            rld * val2.x * val2.x + rld * val2.y * val2.y));
    output[idx] = val2;
  }
  LayerNorm2<T, T2, TPB>(thread_data, hidden, offset, bias, scale, output, eps);
}

template <typename T>
void SkipLayerNormFunctor<T>::operator()(const int num, const int hidden,
                                         const T *input1, const T *input2,
                                         const float *scale, const float *bias,
                                         T *output, T eps,
                                         hipStream_t stream) {
  int block = num / hidden;
  if (hidden <= 32) {
    const int threads = 32;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else if (hidden <= 128) {
    const int threads = 128;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else if (hidden == 384) {
    const int threads = 384;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else {
    const int threads = 256;
    if (hidden % 2 == 0) {
#ifdef SUPPORTS_CUDA_FP16
      if (std::is_same<T, float>::value) {
#endif
        SkipLayerNormKernel2<float, float2,
                             threads><<<block, threads, 0, stream>>>(
            num, hidden / 2, reinterpret_cast<const float2 *>(input1),
            reinterpret_cast<const float2 *>(input2),
            reinterpret_cast<float2 *>(output),
            reinterpret_cast<const float2 *>(scale),
            reinterpret_cast<const float2 *>(bias), eps);
#ifdef SUPPORTS_CUDA_FP16
      } else if (std::is_same<T, __half>::value) {
        SkipLayerNormKernel2<__half, __half2,
                             threads><<<block, threads, 0, stream>>>(
            num, hidden / 2, reinterpret_cast<const __half2 *>(input1),
            reinterpret_cast<const __half2 *>(input2),
            reinterpret_cast<__half2 *>(output),
            reinterpret_cast<const float2 *>(scale),
            reinterpret_cast<const float2 *>(bias), eps);
      } else {
        assert(false);
        // should not be here
      }
#endif
    } else {
      SkipLayerNormKernel<T, threads><<<block, threads, 0, stream>>>(
          num, hidden, input1, input2, output, scale, bias, eps);
    }
  }
}

template class SkipLayerNormFunctor<float>;

#ifdef SUPPORTS_CUDA_FP16
template class SkipLayerNormFunctor<half>;
#endif

}  // namespace math
}  // namespace operators
}  // namespace paddle
