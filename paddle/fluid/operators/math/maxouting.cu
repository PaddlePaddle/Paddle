#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/math/maxouting.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__global__ void KernelMaxOut(const int nthreads, const T* input_data,
                             const int channels, const int input_height,
                             const int input_width, const int groups,
                             const int axis, T* output_data) {
  const int size = input_height * input_width * channels / groups;
  const int feat_len = input_height * input_width;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int batch_idx = i / size;
    int batch_offset = i % size;
    int channel_idx, feat_idx, data_idx;
    if (axis == 1) {
      channel_idx = batch_offset / feat_len;
      feat_idx = batch_offset % feat_len;
      data_idx =
          (batch_idx * size + channel_idx * feat_len) * groups + feat_idx;
    } else {
      channel_idx = batch_offset % channels;
      feat_idx = batch_offset / channels;
      data_idx =
          (batch_idx * size + feat_idx * channels + channel_idx) * groups;
    }
    T ele = static_cast<T>(-FLT_MAX);
    for (int g = 0; g < groups; ++g) {
      int idx_offset = (axis == 1 ? g * feat_len : g);
      T x = input_data[data_idx + idx_offset];
      ele = ele > x ? ele : x;
    }
    output_data[i] = ele;
  }
}
template <typename T>
__global__ void KernelMaxoutGrad(const int nthreads, const T* input_data,
                                 const T* output_data, const T* output_grad,
                                 T* input_grad, const int channels,
                                 const int input_height, const int input_width,
                                 const int groups, const int axis) {
  const int size = input_height * input_width * channels / groups;
  const int feat_len = input_height * input_width;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int batch_idx = i / size;
    int batch_offset = i % size;
    int channel_idx, feat_idx, data_idx;
    if (axis == 1) {
      channel_idx = batch_offset / feat_len;
      feat_idx = batch_offset % feat_len;
      data_idx =
          (batch_idx * size + channel_idx * feat_len) * groups + feat_idx;
    } else {
      channel_idx = batch_offset % channels;
      feat_idx = batch_offset / channels;
      data_idx =
          (batch_idx * size + feat_idx * channels + channel_idx) * groups;
    }
    int max_index = -1;
    bool continue_match = true;
    for (int g = 0; g < groups && continue_match; ++g) {
      int idx_offset = (axis == 1 ? g * feat_len : g);
      if (input_data[data_idx + idx_offset] == output_data[i]) {
        max_index = data_idx + idx_offset;
        continue_match = false;
        break;
      }
    }
    if (max_index != -1) {
      input_grad[max_index] += output_grad[index];
    }
  }
}
/*
 * All tensors are in NCHW or NHWC format.
 */
template <typename T>
class MaxOutFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, framework::Tensor* output,
                  const int groups, const int axis) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[axis];
    const int input_height = (axis == 1 ? input.dims()[2] : input.dims()[1]);
    const int input_width = (axis == 1 ? input.dims()[3] : input.dims()[2]);
    const int output_channels = output->dims()[axis];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(context.GetPlace());
    int nthreads = output->numel();
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxOut<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width, groups,
        axis, output_data);
  }
};
/*
 * All tensors are in NCHW or NHWC format.
 */
template <typename T>
class MaxOutGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& input, framework::Tensor* input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, const int groups,
                  const int axis) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[axis];
    const int input_height = (axis == 1 ? input.dims()[2] : input.dims()[1]);
    const int input_width = (axis == 1 ? input.dims()[3] : input.dims()[2]);
    const int output_channels = output.dims()[axis];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());
    int nthreads = output.numel();
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxoutGrad<T><<<grid, threads, 0, context.stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_height, input_width, groups, axis);
  }
};

template class MaxOutGradFunctor<platform::CUDADeviceContext, float>;
template class MaxOutGradFunctor<platform::CUDADeviceContext, double>;

template class MaxOutFunctor<platform::CUDADeviceContext, float>;
template class MaxOutFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
