/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/uniform_random_op.h"

namespace paddle {
namespace operators {

template <typename T, typename DeviceContext>
class GPUUniformRandomKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    phi::DenseTensor* tensor = nullptr;
    auto out_var = context.OutputVar("Out");
    std::vector<int64_t> new_shape;
    auto list_new_shape_tensor =
        context.MultiInput<phi::DenseTensor>("ShapeTensorList");
    if (list_new_shape_tensor.size() > 0 || context.HasInput("ShapeTensor")) {
      if (context.HasInput("ShapeTensor")) {
        auto* shape_tensor = context.Input<phi::DenseTensor>("ShapeTensor");
        new_shape = GetNewDataFromShapeTensor(shape_tensor);
      } else if (list_new_shape_tensor.size() > 0) {
        new_shape = GetNewDataFromShapeTensorList(list_new_shape_tensor);
      }
    }

    if (out_var->IsType<phi::SelectedRows>()) {
      auto* selected_rows = out_var->GetMutable<phi::SelectedRows>();
      tensor = selected_rows->mutable_value();
      auto shape = context.Attr<std::vector<int64_t>>("shape");
      if (!new_shape.empty()) shape = new_shape;
      tensor->Resize(common::make_ddim(shape));
      selected_rows->mutable_rows()->reserve(shape[0]);
    } else if (out_var->IsType<phi::DenseTensor>()) {
      tensor = out_var->GetMutable<phi::DenseTensor>();
      if (!new_shape.empty()) tensor->Resize(common::make_ddim(new_shape));
    } else {
      PADDLE_THROW(phi::errors::InvalidArgument(
          "Expected type of Output(out) in uniform_random_op must be "
          "phi::DenseTensor, "
          "SelectedRows. But got "
          "unsupport type: %s.",
          framework::ToTypeName(out_var->Type())));
    }
    UniformRandom<T>(context, tensor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
PD_REGISTER_STRUCT_KERNEL(uniform_random_batch_size_like,
                          GPU,
                          ALL_LAYOUT,
                          ops::GPUUniformRandomKernel,
                          float,
                          double) {}
