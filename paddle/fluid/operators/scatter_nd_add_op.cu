/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/gather_op.h"
#include "paddle/fluid/operators/scatter.cu.h"
#include "paddle/fluid/operators/scatter_nd_add_op.h"

namespace paddle {
namespace operators {

template <typename DeviceContext, typename T>
class ScatterNdAddOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "This kernel only runs on GPU device.");
    auto *X = ctx.Input<Tensor>("X");
    auto *Ids = ctx.Input<Tensor>("Index");
    auto *Updates = ctx.Input<Tensor>("Updates");
    auto *Out = ctx.Output<Tensor>("Out");

    framework::TensorCopySync(*X, ctx.GetPlace(), Out);
    const auto &index_type = Ids->type();
    bool index_type_match = index_type == framework::proto::VarType::INT32 ||
                            index_type == framework::proto::VarType::INT64;
    PADDLE_ENFORCE_EQ(
        index_type_match, true,
        "Index holds the wrong type, it holds %s, but desires to be %s or %s",
        paddle::framework::DataTypeToString(index_type),
        paddle::framework::DataTypeToString(framework::proto::VarType::INT32),
        paddle::framework::DataTypeToString(framework::proto::VarType::INT64));
    if (index_type == framework::proto::VarType::INT32) {
      GPUScatterNdAdd<DeviceContext, T, int32_t>(ctx, *Updates, *Ids, Out);
    } else {
      GPUScatterNdAdd<DeviceContext, T, int64_t>(ctx, *Updates, *Ids, Out);
    }
  }
};

template <typename DeviceContext, typename T>
class ScatterNdAddGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      "This kernel only runs on GPU device.");
    auto *dX = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *dUpdates = ctx.Output<Tensor>(framework::GradVarName("Updates"));
    auto *Ids = ctx.Input<Tensor>("Index");
    auto *dOut = ctx.Input<Tensor>(framework::GradVarName("Out"));
    if (dX) {
      // In place gradient: dX = dO
      framework::TensorCopy(*dOut, ctx.GetPlace(), dX);
    }
    if (dUpdates) {
      dUpdates->mutable_data<T>(ctx.GetPlace());
      // Gradient by Gather
      const auto &index_type = Ids->type();
      if (index_type == framework::proto::VarType::INT32) {
        GPUGatherNd<DeviceContext, T, int32_t>(ctx, *dOut, *Ids, dUpdates);
      } else {
        GPUGatherNd<DeviceContext, T, int64_t>(ctx, *dOut, *Ids, dUpdates);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(scatter_nd_add,
                        ops::ScatterNdAddOpCUDAKernel<CUDA, float>,
                        ops::ScatterNdAddOpCUDAKernel<CUDA, double>,
                        ops::ScatterNdAddOpCUDAKernel<CUDA, int64_t>,
                        ops::ScatterNdAddOpCUDAKernel<CUDA, int>,
                        ops::ScatterNdAddOpCUDAKernel<CUDA, plat::float16>);

REGISTER_OP_CUDA_KERNEL(scatter_nd_add_grad,
                        ops::ScatterNdAddGradOpCUDAKernel<CUDA, float>,
                        ops::ScatterNdAddGradOpCUDAKernel<CUDA, double>,
                        ops::ScatterNdAddGradOpCUDAKernel<CUDA, int64_t>,
                        ops::ScatterNdAddGradOpCUDAKernel<CUDA, int>,
                        ops::ScatterNdAddGradOpCUDAKernel<CUDA, plat::float16>);
