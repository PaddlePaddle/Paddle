#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once

#include <string>
#include <vector>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/sample_prob.h"
#include "paddle/fluid/operators/math/softmax.h"
#include "paddle/fluid/operators/sample_logits_op.h"

namespace paddle {
namespace operators {

// UNDERSTAND: something like take_along_axis in numpy.
template <typename T>
__global__ void GPUTakeAlongD1(size_t size, const int batch_size,
                               const int array_slice_size,
                               const int idx_slice_size, const T* p_array,
                               const int64_t* p_index, T* p_value) {
  const auto value_slice_size = idx_slice_size;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  for (; idx < size; idx += step_size) {
    int i = idx / idx_slice_size;
    auto array_index = p_index[idx];
    p_value[idx] = p_array[i * array_slice_size + array_index];
  }
}

// UNDERSTAND: something like put_along_axis in numpy but if there is duplicate
// indices, scatter is done in += way.
template <typename T>
__global__ void GPUPutAlongD1(size_t size, const int batch_size,
                              const int array_slice_size,
                              const int idx_slice_size, T* p_array,
                              const int64_t* p_index, const T* p_value) {
  const auto value_slice_size = idx_slice_size;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  // size == batch_size
  for (; idx < size; idx += step_size) {
    int i = idx;
    for (int j = 0; j < idx_slice_size; ++j) {
      auto array_index = p_index[i * idx_slice_size + j];
      p_array[i * array_slice_size + array_index] +=
          p_value[i * idx_slice_size + j];
    }
  }
}

// UNDERSTAND: set label as 0,1,...,num_true-1
template <typename T>
__global__ void GPUSetLabel(size_t size, const int num_true, int64_t* p_array) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  for (; idx < size; idx += step_size) {
    p_array[idx] = idx % num_true;
  }
}

// UNDERSTAND: compute accidentdal hits from samples and minus corresponding
// logits by a float max, here 1e20
template <typename T>
__global__ void gpu_compute_remove_accidental_hits(const int size,
                                                   const int num_true,
                                                   const int idx_slice_size,
                                                   const int64_t* p_index,
                                                   T* p_value) {
  const auto value_slice_size = idx_slice_size;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = blockDim.x * gridDim.x;

  for (; idx < size; idx += step_size) {
    int i = idx / idx_slice_size;
    if (idx % idx_slice_size < num_true) continue;
    for (int j = 0; j < num_true; ++j) {
      const auto true_idx = i * idx_slice_size + j;
      if (p_index[true_idx] == p_index[idx]) {
        p_value[idx] -= 1e20;
        break;
      }
    }
  }
}

template <typename T>
class SampleLogitsCUDAKernel : public framework::OpKernel<T> {
 public:
  using Tensor = framework::Tensor;
  void Compute(const framework::ExecutionContext& context) const override {
    // get necessary inputs
    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Labels");
    VLOG(3) << "Enter SampleLogitsCUDAKernel";

    // get necessary outputs
    Tensor* samples = context.Output<Tensor>("Samples");
    Tensor* probabilities = context.Output<Tensor>("Probabilities");
    Tensor* sampled_logits = context.Output<Tensor>("SampledLogits");
    Tensor* sampled_labels = context.Output<Tensor>("SampledLabels");

    // shapes
    const auto batch_size = logits->dims()[0];
    const auto num_classes = logits->dims()[1];
    const auto labels_dim = labels->dims();
    const auto num_true = labels_dim[1];
    const auto samples_dim = samples->dims();

    // attrs
    const auto num_samples = context.Attr<int>("num_samples");
    const bool use_customized_samples =
        context.Attr<bool>("use_customized_samples");
    const bool uniq = context.Attr<bool>("uniq");
    const bool remove_accidental_hits =
        context.Attr<bool>("remove_accidental_hits");

    // device contexts
    auto& dev_ctx = context.cuda_device_context();

    // UNDERSTAND: allocate memories for temporaries
    sampled_logits->mutable_data<T>(samples_dim, context.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    set_zero(dev_ctx, sampled_logits, static_cast<T>(0));

    auto sampled_labels_data =
        sampled_labels->mutable_data<int64_t>(labels_dim, context.GetPlace());
    int threads = 512;
    size_t size = batch_size * num_true;
    int grid = (size + threads - 1) / threads;
    GPUSetLabel<
        T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(
        size, num_true, sampled_labels_data);

    if (use_customized_samples) {
      const Tensor* customized_samples =
          context.Input<Tensor>("CustomizedSamples");
      const Tensor* customized_probabilities =
          context.Input<Tensor>("CustomizedProbabilities");
      samples->ShareDataWith(*customized_samples);
      probabilities->ShareDataWith(*customized_probabilities);
    } else {
      samples->mutable_data<int64_t>(context.GetPlace());
      probabilities->mutable_data<T>(samples_dim, context.GetPlace());
      // UNDERSTAND: sampling
      const auto seed = context.Attr<int>("seed");
      auto sampler_with_prob = math::GPUSampleWithProb<T>();
      sampler_with_prob(context.cuda_device_context(), seed, num_classes, uniq,
                        num_samples, labels, samples, probabilities);
    }

    // UNDERSTAND: gather sampled logits and remove accidental hits if needed
    const auto num_take = samples->dims()[1];
    const auto array_dims = logits->dims();
    const auto idx_dims = samples->dims();

    const T* p_array = logits->data<T>();
    const int64_t* p_index = samples->data<int64_t>();
    T* p_value = sampled_logits->data<T>();

    // src slice size
    const auto array_slice_size = array_dims[1];
    // index slice size
    const auto idx_slice_size = idx_dims[1];

    size = batch_size * num_take;
    grid = (size + threads - 1) / threads;
    GPUTakeAlongD1<
        T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(
        size, batch_size, array_slice_size, idx_slice_size, p_array, p_index,
        p_value);

    if (remove_accidental_hits) {
      const size_t size = batch_size * (num_true + num_samples);
      int grid = (size + threads - 1) / threads;
      gpu_compute_remove_accidental_hits<
          T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(
          size, num_true, idx_slice_size, p_index, p_value);
    }

    // subtracted sampled logits with logQ(y|x)
    auto probs = EigenMatrix<T>::From(*probabilities);
    auto smp_logits = EigenMatrix<T>::From(*sampled_logits);
    smp_logits.device(*dev_ctx.eigen_device()) =
        (smp_logits - probs.log().unaryExpr(TolerableValue<T>()))
            .unaryExpr(TolerableValue<T>());
  }
};

template <typename T>
class SampleLogitsGradCUDAKernel : public framework::OpKernel<T> {
 public:
  using Tensor = framework::Tensor;
  void Compute(const framework::ExecutionContext& context) const override {
    auto logits_grad = context.Output<Tensor>(framework::GradVarName("Logits"));
    const Tensor* samples = context.Input<Tensor>("Samples");
    const Tensor* sampled_logits_grad =
        context.Input<Tensor>(framework::GradVarName("SampledLogits"));
    logits_grad->mutable_data<T>(context.GetPlace());

    auto& dev_ctx = context.cuda_device_context();
    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    set_zero(dev_ctx, logits_grad, static_cast<T>(0));

    // UNDERSTAND: scatter it back to logit_grad
    const auto batch_size = samples->dims()[0];
    const auto num_put = samples->dims()[1];
    const auto array_dims = logits_grad->dims();
    const auto idx_dims = samples->dims();

    T* p_array = logits_grad->data<T>();
    const int64_t* p_index = samples->data<int64_t>();
    const T* p_value = sampled_logits_grad->data<T>();

    // src slice size
    const auto array_slice_size = array_dims[1];
    // index slice size
    const auto idx_slice_size = idx_dims[1];

    int threads = 128;
    const size_t size = batch_size;
    int grid = (size + threads - 1) / threads;

    GPUPutAlongD1<
        T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(
        size, batch_size, array_slice_size, idx_slice_size, p_array, p_index,
        p_value);
  }
};

}  // namespace operators
}  // namespace paddle
namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(sample_logits, ops::SampleLogitsCUDAKernel<float>,
                        ops::SampleLogitsCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(sample_logits_grad,
                        ops::SampleLogitsGradCUDAKernel<float>,
                        ops::SampleLogitsGradCUDAKernel<double>);
