#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/operators/label_smooth_op.h"
namespace paddle {
namespace operators {

template <typename T>
__global__ void LabelSmoothRunOriginKernel(const int N, const float epsilon,
                                           const int label_dim, const T* src,
                                           T* dst) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    dst[idx] = static_cast<T>(1 - epsilon) * src[idx] +
               static_cast<T>(epsilon / label_dim);
  }
}

template <typename T>
__global__ void LabelSmoothRunDistKernel(const int N, const float epsilon,
                                         const int dist_numel, const T* src,
                                         const T* dist_data, T* dst) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    int dist_idx = idx - (idx / dist_numel) * dist_numel;
    dst[idx] = static_cast<T>(1 - epsilon) * src[idx] +
               static_cast<T>(epsilon) * dist_data[dist_idx];
  }
}

template <typename T>
__global__ void LabelSmoothGradRunKernel(const int N, const float epsilon,
                                         const T* src, T* dst) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    dst[idx] = static_cast<T>(1 - epsilon) * src[idx];
  }
}

template <typename DeviceContext, typename T>
class LabelSmoothGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto* out_t = ctx.Output<framework::LoDTensor>("Out");
    auto* in_t = ctx.Input<framework::LoDTensor>("X");
    auto* dist_t = ctx.Input<framework::Tensor>("PriorDist");
    auto label_dim = in_t->dims()[1];
    auto epsilon = ctx.Attr<float>("epsilon");
    auto& dev = *ctx.template device_context<DeviceContext>().eigen_device();
    auto size_prob = in_t->numel();
    const T* in_data = in_t->data<T>();
    T* out_data = out_t->mutable_data<T>(ctx.GetPlace());
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = ctx.cuda_device_context().stream();
    if (dist_t) {
      auto dist_numel = dist_t->numel();
      const T* dist_data = dist_t->data<T>();
      LabelSmoothRunDistKernel<T><<<grid, threads, 0, stream>>>(
          size_prob, epsilon, dist_numel, in_data, dist_data, out_data);

    } else {
      LabelSmoothRunOriginKernel<T><<<grid, threads, 0, stream>>>(
          size_prob, epsilon, label_dim, in_data, out_data);
    }
  }
};

template <typename DeviceContext, typename T>
class LabelSmoothGradGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto* d_out_t = ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* d_in_t = ctx.Output<framework::Tensor>(framework::GradVarName("X"));
    d_in_t->mutable_data<T>(ctx.GetPlace());

    auto epsilon = ctx.Attr<float>("epsilon");
    auto& dev = *ctx.template device_context<DeviceContext>().eigen_device();
    const T* in_data = d_out_t->data<T>();
    auto size_prob = d_out_t->numel();
    T* out_data = d_in_t->mutable_data<T>(ctx.GetPlace());
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = ctx.cuda_device_context().stream();
    LabelSmoothGradRunKernel<T><<<grid, threads, 0, stream>>>(
        size_prob, epsilon, in_data, out_data);
  }
};
}  // namespace operators
}  // namespace paddle
namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    label_smooth,
    ops::LabelSmoothGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LabelSmoothGPUKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    label_smooth_grad,
    ops::LabelSmoothGradGPUKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LabelSmoothGradGPUKernel<paddle::platform::CUDADeviceContext, double>);
