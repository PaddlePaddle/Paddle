#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <tuple>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/diag_v2_op.h"

namespace paddle {
namespace operators {

// Extract the diagonal of a matrix 'x' to a vector 'out'.
template <typename T>
__global__ void ExtractDiagonalKernel(T* out, const T* x, std::ptrdiff_t start,
                                      std::ptrdiff_t size,
                                      const std::ptrdiff_t sumStride,
                                      const std::ptrdiff_t outStride) {
  for (std::ptrdiff_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    const std::ptrdiff_t xOffset = start + sumStride * idx;
    out[outStride * idx] = x[xOffset];
  }
}

// Paste a vector 'x' to the diagonal of a matrix 'out'
template <typename T>
__global__ void PasteDiagonalKernel(T* out, const T* x, std::ptrdiff_t start,
                                    std::ptrdiff_t x_length,
                                    const std::ptrdiff_t sumStride,
                                    const std::ptrdiff_t xStride) {
  for (std::ptrdiff_t idx = blockIdx.x * blockDim.x + threadIdx.x;
       idx < x_length; idx += gridDim.x * blockDim.x) {
    const std::ptrdiff_t outOffset = start + sumStride * idx;
    out[outOffset] = x[xStride * idx];
  }
}

template <typename DeviceContext, typename T>
class DiagV2CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* X = context.Input<framework::Tensor>("X");
    auto* x_data = X->data<T>();
    auto x_dims = X->dims();
    int offset = context.Attr<int>("offset");
    auto* out = context.Output<framework::Tensor>("Out");
    T* out_data = out->mutable_data<T>(context.GetPlace());
    auto out_dims = out->dims();
    auto& dev_ctx = context.template device_context<DeviceContext>();

    auto GetBlockGridSize = [&dev_ctx](int64_t size) {
      const int64_t block_size =
          std::min(size, static_cast<int64_t>(dev_ctx.GetMaxThreadsPerBlock()));
      int64_t max_threads = dev_ctx.GetMaxPhysicalThreadCount();
      const int64_t max_blocks = std::max(((max_threads - 1) / block_size + 1),
                                          static_cast<int64_t>(1));
      const int64_t grid_size =
          std::min(max_blocks, (size + block_size - 1) / block_size);
      return std::tuple<int64_t, int64_t>{block_size, grid_size};
    };

    if (x_dims.size() == 1) {
      float padding_value = context.Attr<float>("padding_value");
      math::SetConstant<DeviceContext, T> set_padding_value;
      set_padding_value(dev_ctx, out, static_cast<T>(padding_value));

      auto x_length = x_dims[0];
      auto size = (offset > 0) ? x_length + offset : x_length - offset;
      const int& x_stride = ComputeStride(0, x_dims);
      if (size > 0) {
        const auto& out_stride_0 = ComputeStride(0, out_dims);
        const auto& out_stride_1 = ComputeStride(1, out_dims);
        auto start =
            (offset >= 0 ? offset * out_stride_1 : -offset * out_stride_0);

        std::tuple<int64_t, int64_t> block_grid_size = GetBlockGridSize(size);

        PasteDiagonalKernel<
            T><<<std::get<1>(block_grid_size), std::get<0>(block_grid_size), 0,
                 dev_ctx.stream()>>>(out_data, x_data, start, x_length,
                                     out_stride_0 + out_stride_1, x_stride);
      }
    } else {
      const int& x_stride_0 = ComputeStride(0, x_dims);
      const int& x_stride_1 = ComputeStride(1, x_dims);

      int64_t size;
      if (offset > 0) {
        size = std::min(x_dims[0], x_dims[1] - offset);
      } else {
        size = std::min(x_dims[0] + offset, x_dims[1]);
      }

      if (size > 0) {
        auto start = (offset >= 0 ? offset * x_stride_1 : -offset * x_stride_0);
        const auto& out_stride_0 = ComputeStride(0, out_dims);

        std::tuple<int64_t, int64_t> block_grid_size = GetBlockGridSize(size);

        ExtractDiagonalKernel<
            T><<<std::get<1>(block_grid_size), std::get<0>(block_grid_size), 0,
                 dev_ctx.stream()>>>(out_data, x_data, start, size,
                                     x_stride_0 + x_stride_1, out_stride_0);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    diag_v2, ops::DiagV2CUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::DiagV2CUDAKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::DiagV2CUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::DiagV2CUDAKernel<paddle::platform::CUDADeviceContext, double>);
