#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/operators/graph_send_recv_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/place.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T, typename IndexT>
struct GraphSendRecvSumCUDAFunctor {
  DEVICE inline void operator()(const T* params, T* output, const IndexT& in_i,
                                const IndexT& out_i) {
    paddle::platform::CudaAtomicAdd(output + out_i, *(params + in_i));
  }
};

template <typename T, typename IndexT>
struct GraphSendRecvMaxCUDAFunctor {
  DEVICE inline void operator()(const T* params, T* output, const IndexT& in_i,
                                const IndexT& out_i) {
    paddle::platform::CudaAtomicMax(output + out_i, *(params + in_i));
  }
};

template <typename T, typename IndexT>
struct GraphSendRecvMinCUDAFunctor {
  DEVICE inline void operator()(const T* params, T* output, const IndexT& in_i,
                                const IndexT& out_i) {
    paddle::platform::CudaAtomicMin(output + out_i, *(params + in_i));
  }
};

template <typename T, typename IndexT, typename Functor>
__global__ void GraphSendRecvCUDAKernel(const T* params,
                                        const IndexT* src_indices,
                                        const IndexT* dst_indices, T* output,
                                        size_t index_size, size_t slice_size,
                                        Functor functor) {
  CUDA_KERNEL_LOOP_TYPE(i, index_size * slice_size, int64_t) {
    int64_t indices_i = i / slice_size;
    int64_t slice_i = i - indices_i * slice_size;
    IndexT src_i = src_indices[indices_i];
    IndexT dst_i = dst_indices[indices_i];
    int64_t in_i = src_i * slice_size + slice_i;
    int64_t out_i = dst_i * slice_size + slice_i;
    functor(params, output, in_i, out_i);
  }
}

// For max
template <typename T>
__global__ void InputResetMaxCUDAKernel(T* output, size_t input_size,
                                        size_t slice_size) {
  CUDA_KERNEL_LOOP_TYPE(i, input_size * slice_size, int64_t) {
    if (*(output + i) == std::numeric_limits<T>::min()) {
      *(output + i) = 0;
    }
  }
}

// For min
template <typename T>
__global__ void InputResetMinCUDAKernel(T* output, size_t input_size,
                                        size_t slice_size) {
  CUDA_KERNEL_LOOP_TYPE(i, input_size * slice_size, int64_t) {
    if (*(output + i) == std::numeric_limits<T>::max()) {
      *(output + i) = 0;
    }
  }
}

// Get dst_count
template <typename T, typename IndexT>
__global__ void ComputeCountCUDAKernel(int* count, const IndexT* dst_indices,
                                       size_t index_size) {
  CUDA_KERNEL_LOOP_TYPE(i, index_size, int64_t) {
    IndexT dst_i = dst_indices[i];
    paddle::platform::CudaAtomicAdd(count + dst_i, 1);
  }
}

// For forward mean
template <typename T>
__global__ void ManipulateMeanCUDAKernel(T* output, int* count,
                                         size_t input_size, size_t slice_size) {
  CUDA_KERNEL_LOOP_TYPE(i, input_size * slice_size, int64_t) {
    int64_t c_index = i / slice_size;
    if (*(count + c_index) > 1) {
      *(output + i) = *(output + i) / *(count + c_index);
    }
  }
}

// For backward mean
template <typename T, typename IndexT>
__global__ void ManipulateMeanGradCUDAKernel(
    const T* params, const IndexT* src_indices, const IndexT* dst_indices,
    T* output, size_t index_size, size_t slice_size, const int* dst_count) {
  CUDA_KERNEL_LOOP_TYPE(i, index_size * slice_size, int64_t) {
    int64_t indices_i = i / slice_size;
    int64_t slice_i = i - indices_i * slice_size;
    IndexT src_i = src_indices[indices_i];
    IndexT dst_i = dst_indices[indices_i];
    int64_t in_i = src_i * slice_size + slice_i;
    int64_t out_i = dst_i * slice_size + slice_i;
    paddle::platform::CudaAtomicAdd(output + out_i,
                                    *(params + in_i) / dst_count[src_i]);
  }
}

// For backward min and max
template <typename T, typename IndexT>
__global__ void ManipulateMinMaxGradCUDAKernel(
    const T* params, const IndexT* src_indices, const IndexT* dst_indices,
    T* output, size_t index_size, size_t slice_size, const T* ptr_input,
    const T* ptr_output) {
  CUDA_KERNEL_LOOP_TYPE(i, index_size * slice_size, int64_t) {
    int64_t indices_i = i / slice_size;
    int64_t slice_i = i - indices_i * slice_size;
    IndexT src_i = src_indices[indices_i];
    IndexT dst_i = dst_indices[indices_i];
    int64_t in_i = src_i * slice_size + slice_i;
    int64_t out_i = dst_i * slice_size + slice_i;
    paddle::platform::CudaAtomicAdd(
        output + out_i,
        *(params + in_i) * (*(ptr_input + out_i) == *(ptr_output + in_i)));
  }
}

template <typename DeviceContext, typename T, typename IndexT>
void GraphSendRecvOpCUDAKernelLaunchHelper(
    const framework::ExecutionContext& ctx, const Tensor& src_index,
    const Tensor& dst_index) {
  auto* X = ctx.Input<Tensor>("X");
  auto* Y = ctx.Output<Tensor>("Out");
  std::string pool_type = ctx.Attr<std::string>("pool_type");

  const int& index_size = src_index.dims()[0];

  T* p_output = Y->mutable_data<T>(ctx.GetPlace());
  const auto& src_dims = X->dims();
  int64_t memset_size = 1;
  for (int i = 0; i < src_dims.size(); ++i) {
    memset_size *= src_dims[i];
  }
  const size_t& memset_bytes = memset_size * sizeof(T);
  if (pool_type == "SUM" || pool_type == "MEAN") {
#ifdef PADDLE_WITH_HIP
    hipMemset(p_output, 0, memset_bytes);
#else
    hipMemset(p_output, 0, memset_bytes);
#endif
  } else if (pool_type == "MAX") {
    thrust::device_ptr<T> p_output_ptr(p_output);
    thrust::fill(thrust::device, p_output_ptr, p_output_ptr + memset_size,
                 std::numeric_limits<T>::min());
  } else if (pool_type == "MIN") {
    thrust::device_ptr<T> p_output_ptr(p_output);
    thrust::fill(thrust::device, p_output_ptr, p_output_ptr + memset_size,
                 std::numeric_limits<T>::max());
  }

  if (index_size == 0) return;

  int64_t slice_size = 1;
  for (int i = 1; i < src_dims.size(); ++i) {
    slice_size *= src_dims[i];
  }
  const T* p_src = X->data<T>();
  const IndexT* s_index = src_index.data<IndexT>();
  const IndexT* d_index = dst_index.data<IndexT>();

#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int64_t n = slice_size * index_size;
  const auto& dev_ctx = ctx.cuda_device_context();
  int64_t max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int64_t grid_tmp = (n + block - 1) / block;
  int64_t grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  int64_t input_size = src_dims[0];
  if (pool_type == "SUM") {
    GraphSendRecvSumCUDAFunctor<T, IndexT> functor;
    GraphSendRecvCUDAKernel<T, IndexT,
                            GraphSendRecvSumCUDAFunctor<T, IndexT>><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, functor);
  } else if (pool_type == "MAX") {
    GraphSendRecvMaxCUDAFunctor<T, IndexT> functor;
    GraphSendRecvCUDAKernel<T, IndexT,
                            GraphSendRecvMaxCUDAFunctor<T, IndexT>><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, functor);

    int64_t grid_max_tmp = (input_size * slice_size + block - 1) / block;
    int64_t grid_max =
        grid_max_tmp < max_grid_dimx ? grid_max_tmp : max_grid_dimx;
    InputResetMaxCUDAKernel<
        T><<<grid_max, block, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(
                 ctx.device_context())
                 .stream()>>>(p_output, input_size, slice_size);
  } else if (pool_type == "MIN") {
    GraphSendRecvMinCUDAFunctor<T, IndexT> functor;
    GraphSendRecvCUDAKernel<T, IndexT,
                            GraphSendRecvMinCUDAFunctor<T, IndexT>><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, functor);

    int64_t grid_min_tmp = (input_size * slice_size + block - 1) / block;
    int64_t grid_min =
        grid_min_tmp < max_grid_dimx ? grid_min_tmp : max_grid_dimx;
    InputResetMinCUDAKernel<
        T><<<grid_min, block, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(
                 ctx.device_context())
                 .stream()>>>(p_output, input_size, slice_size);
  } else if (pool_type == "MEAN") {
    GraphSendRecvSumCUDAFunctor<T, IndexT> functor;
    GraphSendRecvCUDAKernel<T, IndexT,
                            GraphSendRecvSumCUDAFunctor<T, IndexT>><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, functor);

    auto* dst_count = ctx.Output<Tensor>("Dst_count");
    int* p_dst_count = dst_count->mutable_data<int>(ctx.GetPlace());

#ifdef PADDLE_WITH_HIP
    hipMemset(p_dst_count, 0, input_size * sizeof(int));
#else
    hipMemset(p_dst_count, 0, input_size * sizeof(int));
#endif

    int64_t grid_count = (index_size + block - 1) / block;
    ComputeCountCUDAKernel<
        T, IndexT><<<grid_count, block, 0,
                     reinterpret_cast<const platform::CUDADeviceContext&>(
                         ctx.device_context())
                         .stream()>>>(p_dst_count, d_index, index_size);

    int64_t grid_mean_tmp = (input_size * slice_size + block - 1) / block;
    int64_t grid_mean =
        grid_mean_tmp < max_grid_dimx ? grid_mean_tmp : max_grid_dimx;
    ManipulateMeanCUDAKernel<
        T><<<grid_mean, block, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(
                 ctx.device_context())
                 .stream()>>>(p_output, p_dst_count, input_size, slice_size);
  }
}

template <typename DeviceContext, typename T, typename IndexT>
void GraphSendRecvGradOpCUDAKernelLaunchHelper(
    const framework::ExecutionContext& ctx, const Tensor& src_index,
    const Tensor& dst_index) {
  auto* X = ctx.Input<Tensor>(framework::GradVarName("Out"));
  auto* Y = ctx.Output<Tensor>(framework::GradVarName("X"));
  std::string pool_type = ctx.Attr<std::string>("pool_type");

  const int& index_size = src_index.dims()[0];

  T* p_output = Y->mutable_data<T>(ctx.GetPlace());
  const auto& src_dims = X->dims();
  int64_t memset_size = 1;
  for (int i = 0; i < src_dims.size(); ++i) {
    memset_size *= src_dims[i];
  }
  const size_t& memset_bytes = memset_size * sizeof(T);

#ifdef PADDLE_WITH_HIP
  hipMemset(p_output, 0, memset_bytes);
#else
  hipMemset(p_output, 0, memset_bytes);
#endif

  if (index_size == 0) return;

  int64_t slice_size = 1;
  for (int i = 1; i < src_dims.size(); ++i) {
    slice_size *= src_dims[i];
  }
  const T* p_src = X->data<T>();
  const IndexT* s_index = src_index.data<IndexT>();
  const IndexT* d_index = dst_index.data<IndexT>();

#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int64_t n = slice_size * index_size;
  const auto& dev_ctx = ctx.cuda_device_context();
  int64_t max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int64_t grid_tmp = (n + block - 1) / block;
  int64_t grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  int64_t input_size = src_dims[0];
  if (pool_type == "SUM") {
    GraphSendRecvSumCUDAFunctor<T, IndexT> functor;
    GraphSendRecvCUDAKernel<T, IndexT,
                            GraphSendRecvSumCUDAFunctor<T, IndexT>><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, functor);
  } else if (pool_type == "MEAN") {
    auto* dst_count = ctx.Input<Tensor>("Dst_count");
    const int* s_count = dst_count->data<int>();
    ManipulateMeanGradCUDAKernel<T, IndexT><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, s_count);
  } else if (pool_type == "MAX" || pool_type == "MIN") {
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Input<Tensor>("Out");
    const T* ptr_input = input->data<T>();
    const T* ptr_output = output->data<T>();
    ManipulateMinMaxGradCUDAKernel<T, IndexT><<<
        grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                            ctx.device_context())
                            .stream()>>>(p_src, s_index, d_index, p_output,
                                         index_size, slice_size, ptr_input,
                                         ptr_output);
  }
}

template <typename DeviceContext, typename T>
class GraphSendRecvOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* src_index = ctx.Input<Tensor>("Src_index");
    auto* dst_index = ctx.Input<Tensor>("Dst_index");
    auto index_type = framework::TransToProtoVarType(src_index->dtype());

    if (index_type == framework::proto::VarType::INT32) {
      GraphSendRecvOpCUDAKernelLaunchHelper<DeviceContext, T, int>(
          ctx, *src_index, *dst_index);
    } else if (index_type == framework::proto::VarType::INT64) {
      GraphSendRecvOpCUDAKernelLaunchHelper<DeviceContext, T, int64_t>(
          ctx, *src_index, *dst_index);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported Src_index or Dst_index dtype, expected int, int64, but "
          "got %s.",
          index_type));
    }
  }
};

template <typename DeviceContext, typename T>
class GraphSendRecvGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* src_index = ctx.Input<Tensor>("Dst_index");
    auto* dst_index = ctx.Input<Tensor>("Src_index");
    auto index_type = framework::TransToProtoVarType(src_index->dtype());

    if (index_type == framework::proto::VarType::INT32) {
      GraphSendRecvGradOpCUDAKernelLaunchHelper<DeviceContext, T, int>(
          ctx, *src_index, *dst_index);
    } else if (index_type == framework::proto::VarType::INT64) {
      GraphSendRecvGradOpCUDAKernelLaunchHelper<DeviceContext, T, int64_t>(
          ctx, *src_index, *dst_index);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported Src_index or Dst_index dtype, expected int, int64, but "
          "got %s.",
          index_type));
    }
  }
};

}  // namespace operators
}  // namespace paddle

using CUDA = paddle::platform::CUDADeviceContext;
namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(graph_send_recv,
                        ops::GraphSendRecvOpCUDAKernel<CUDA, float>,
                        ops::GraphSendRecvOpCUDAKernel<CUDA, double>,
                        ops::GraphSendRecvOpCUDAKernel<CUDA, int>,
                        ops::GraphSendRecvOpCUDAKernel<CUDA, int64_t>);

REGISTER_OP_CUDA_KERNEL(graph_send_recv_grad,
                        ops::GraphSendRecvGradOpCUDAKernel<CUDA, float>,
                        ops::GraphSendRecvGradOpCUDAKernel<CUDA, double>,
                        ops::GraphSendRecvGradOpCUDAKernel<CUDA, int>,
                        ops::GraphSendRecvGradOpCUDAKernel<CUDA, int64_t>);
