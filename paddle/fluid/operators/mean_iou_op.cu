#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/mean_iou_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void CountCUDAKernel(const int num_classes, const int count,
                                const T* predictions, const T* labels,
                                int* wrong, int* correct) {
  extern __shared__ int blcok_cache[];
  int* wrong_c = blcok_cache;
  int* correct_c = blcok_cache + num_classes;
  // init cache
  for (int i = threadIdx.x; i < num_classes * 2; i += blockDim.x) {
    blcok_cache[i] = 0;
  }
  __syncthreads();

  T pred;
  T label;
  CUDA_1D_KERNEL_LOOP(i, count) {
    pred = predictions[i];
    label = labels[i];
    if (pred == label) {
      atomicAdd(correct_c + pred, 1);
    } else {
      atomicAdd(wrong_c + pred, 1);
      atomicAdd(wrong_c + label, 1);
    }
  }

  __syncthreads();

  for (int i = threadIdx.x; i < num_classes; i += blockDim.x) {
    atomicAdd(wrong + i, wrong_c[i]);
    atomicAdd(correct + i, correct_c[i]);
  }
}

__global__ void ComputeIoUCUDAKernel(const int num_classes, int* wrong,
                                     int* correct, float* ious, float* iou) {
  __shared__ int valid_count_c;
  if (threadIdx.x == 0) {
    valid_count_c = 0;
  }
  __syncthreads();
  CUDA_1D_KERNEL_LOOP(i, num_classes) {
    int wrong_n = wrong[i];
    int correct_n = correct[i];
    int denominator = wrong_n + correct_n;
    if (denominator > 0) {
      atomicAdd(&valid_count_c, 1);
      ious[i] = static_cast<float>(correct_n) / denominator;
    } else {
      ious[i] = 0;
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    float iou_sum = 0;
    for (int i = 0; i < num_classes; ++i) {
      iou_sum += ious[i];
    }
    iou[0] += iou_sum / valid_count_c;
  }
}

template <typename T>
class MeanIoUCUDAOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto& place = *dev_ctx.eigen_device();
    // get input and output tensor
    auto* predictions = ctx.Input<Tensor>("Predictions");
    auto* labels = ctx.Input<Tensor>("Labels");
    auto* out_mean_iou = ctx.Output<Tensor>("OutMeanIou");
    auto* out_wrong = ctx.Output<Tensor>("OutWrong");
    auto* out_correct = ctx.Output<Tensor>("OutCorrect");
    int num_classes = static_cast<int>(ctx.Attr<int>("num_classes"));

    // Get data ptr
    const T* predictions_data = predictions->data<T>();
    const T* labels_data = labels->data<T>();
    int* out_wrong_data = out_wrong->mutable_data<int>(ctx.GetPlace());
    int* out_correct_data = out_correct->mutable_data<int>(ctx.GetPlace());
    float* out_mean_iou_data =
        out_mean_iou->mutable_data<float>(ctx.GetPlace());

    // Get Eigen tensor
    auto out_mean_iou_t = EigenTensor<float, 1>::From(*out_mean_iou);
    auto out_wrong_t = EigenTensor<int, 1>::From(*out_wrong);
    auto out_correct_t = EigenTensor<int, 1>::From(*out_correct);

    // Temporary memory
    auto tmp_ious_data = memory::Alloc(dev_ctx, num_classes * sizeof(float));
    float* ious_data = static_cast<float*>(tmp_ious_data->ptr());

    // Init out_wrong, out_correct and out_mean_iou
    out_wrong_t.device(place) = out_wrong_t.constant(0);
    out_correct_t.device(place) = out_correct_t.constant(0);
    out_mean_iou_t.device(place) = out_mean_iou_t.constant(0.0f);

    // collect pre wrong, correct and mean_iou
    auto in_mean_ious = ctx.MultiInput<Tensor>("InMeanIou");
    for (int i = 0; i < in_mean_ious.size(); ++i) {
      out_mean_iou_t.device(place) +=
          EigenTensor<float, 1>::From(*in_mean_ious[i]);
    }
    auto in_wrongs = ctx.MultiInput<Tensor>("InWrongs");
    for (int i = 0; i < in_wrongs.size(); ++i) {
      out_wrong_t.device(place) += EigenTensor<int, 1>::From(*in_wrongs[i]);
    }
    auto in_corrects = ctx.MultiInput<Tensor>("InCorrects");
    for (int i = 0; i < in_corrects.size(); ++i) {
      out_correct_t.device(place) += EigenTensor<int, 1>::From(*in_corrects[i]);
    }
    // compute
    auto stream = ctx.cuda_device_context().stream();
    int block = PADDLE_CUDA_NUM_THREADS;
    int grid = (predictions->numel() + block - 1) / block;
    int cache_size = (num_classes * 2 + 1) * sizeof(int);
    CountCUDAKernel<T><<<grid, block, cache_size, stream>>>(
        num_classes, predictions->numel(), predictions_data, labels_data,
        out_wrong_data, out_correct_data);

    ComputeIoUCUDAKernel<<<1, block, 0, stream>>>(num_classes, out_wrong_data,
                                                  out_correct_data, ious_data,
                                                  out_mean_iou_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(mean_iou, ops::MeanIoUCUDAOpKernel<int>,
                        ops::MeanIoUCUDAOpKernel<int64_t>,
                        ops::MeanIoUCUDAOpKernel<int32_t>);
