#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/lookup_table_op.h"
#include "paddle/fluid/platform/assert.h"
#include "paddle/fluid/platform/cuda_primitives.h"

#define CLOG std::cout

namespace paddle {
namespace operators {

struct Formater {
  std::string message;
  std::string name;
  std::vector<int> dims;
  std::type_index dtype{typeid(const char)};
  framework::LoD lod;
  int summarize;
  void* data{nullptr};

  void operator()(size_t size) {
    // PrintMessage();
    // PrintName();
    // PrintDims();
    // PrintDtype();
    // PrintLod();
    PrintData(size);
  }

 private:
  void PrintMessage() { CLOG << std::time(nullptr) << "\t" << message << "\t"; }
  void PrintName() {
    if (!name.empty()) {
      CLOG << "Tensor[" << name << "]" << std::endl;
    }
  }
  void PrintDims() {
    if (!dims.empty()) {
      CLOG << "\tshape: [";
      for (auto i : dims) {
        CLOG << i << ",";
      }
      CLOG << "]" << std::endl;
    }
  }
  void PrintDtype() {
    if (dtype.hash_code() != typeid(const char).hash_code()) {
      CLOG << "\tdtype: " << dtype.name() << std::endl;
    }
  }
  void PrintLod() {
    if (!lod.empty()) {
      CLOG << "\tLoD: [";
      for (auto level : lod) {
        CLOG << "[ ";
        for (auto i : level) {
          CLOG << i << ",";
        }
        CLOG << " ]";
      }
      CLOG << "]" << std::endl;
    }
  }

  void PrintData(size_t size) {
    PADDLE_ENFORCE_NOT_NULL(data);
    // print float
    if (dtype.hash_code() == typeid(const float).hash_code()) {
      Display<float>(size);
    } else if (dtype.hash_code() == typeid(const double).hash_code()) {
      Display<double>(size);
    } else if (dtype.hash_code() == typeid(const int).hash_code()) {
      Display<int>(size);
    } else if (dtype.hash_code() == typeid(const int64_t).hash_code()) {
      Display<int64_t>(size);
    } else if (dtype.hash_code() == typeid(const bool).hash_code()) {
      Display<bool>(size);
    } else {
      CLOG << "\tdata: unprintable type: " << dtype.name() << std::endl;
    }
  }

  template <typename T>
  void Display(size_t size) {
    auto* d = reinterpret_cast<T*>(data);
    CLOG << "\tdata: " << size << std::endl;
    if (summarize != -1) {
      summarize = 10000;
      CLOG << "Value of summarize = " << summarize << std::endl;
      for (int i = 0; i < summarize; i++) {
        CLOG << d[i] << ",";
      }
    } else {
      for (size_t i = 0; i < size; i++) {
        CLOG << d[i] << ",";
      }
    }
    CLOG << std::endl;
  }
};

template <typename T, int BlockDimX, int BlockDimY, int GridDimX,
          bool PaddingFlag>
__global__ void LookupTable(T* output, const T* table, const int64_t* ids,
                            const int64_t N, const int64_t K, const int64_t D,
                            const int64_t padding_idx) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int64_t id = ids[idy];
    PADDLE_ASSERT(id >= 0);
    PADDLE_ASSERT(id < N);
    T* out = output + idy * D;
    const T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      if (PaddingFlag) {
        if (id == padding_idx)
          out[i] = static_cast<T>(0);
        else
          out[i] = tab[i];
      } else {
        out[i] = tab[i];
      }
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void LookupTableGrad(T* table, const T* output, const int64_t* ids,
                                const int64_t N, const int64_t K,
                                const int64_t D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int id = ids[idy];
    PADDLE_ASSERT(id >= 0);
    PADDLE_ASSERT(id < N);
    const T* out = output + idy * D;
    T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      tab[i] = tab[i] + out[i];
      // paddle::platform::CudaAtomicAdd(&tab[i], out[i]);
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T>
class LookupTableCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* table_t = context.Input<LoDTensor>("W");
    int64_t padding_idx = context.Attr<int64_t>("padding_idx");
    auto* ids_var = context.InputVar("Ids");
    Tensor* output_t = context.Output<Tensor>("Out");

    framework::LoDTensor print_tensor_table;
    print_tensor_table.set_lod(table_t->lod());
    print_tensor_table.Resize(table_t->dims());

    if (paddle::platform::is_cpu_place(table_t->place())) {
      print_tensor_table.ShareDataWith(*table_t);
    } else {
      // copy data to cpu to print
      paddle::platform::CPUPlace place;
      framework::TensorCopy(*table_t, place, &print_tensor_table);
    }
    Formater formater1;
    formater1.dtype = print_tensor_table.type();
    formater1.data = reinterpret_cast<void*>(print_tensor_table.data<void>());
    // formater1(print_tensor_table.numel());

    int64_t* ids;
    int64_t K;

    // The type of Ids(Input) is SelectedRows or LoDTensor, when Ids's type
    // is LoDTensor, this tensor contains the ids to be looked up in W;
    // when Ids's type is SelectedRows, the rows of Ids contains the
    // ids to be looked up in W.
    if (ids_var->IsType<framework::LoDTensor>()) {
      auto* ids_t = context.Input<LoDTensor>("Ids");
      ids = const_cast<int64_t*>(ids_t->data<int64_t>());
      K = ids_t->numel();

      framework::LoDTensor print_tensor_ids;
      print_tensor_ids.set_lod(ids_t->lod());
      print_tensor_ids.Resize(ids_t->dims());

      if (paddle::platform::is_cpu_place(ids_t->place())) {
        print_tensor_ids.ShareDataWith(*ids_t);
      } else {
        // copy data to cpu to print
        paddle::platform::CPUPlace place;
        framework::TensorCopy(*ids_t, place, &print_tensor_ids);
      }
      Formater formater2;
      formater2.dtype = print_tensor_ids.type();
      formater2.data = reinterpret_cast<void*>(print_tensor_ids.data<void>());
      // formater2(print_tensor_ids.numel());

    } else if (ids_var->IsType<framework::SelectedRows>()) {
      // std::cout << "Oh this is dealing with selected rows" << std::endl;
      auto* ids_t = context.Input<framework::SelectedRows>("Ids");
      ids = const_cast<int64_t*>(ids_t->rows().CUDAData(context.GetPlace()));
      K = ids_t->rows().size();
      output_t->Resize({K, table_t->dims()[1]});
    } else {
      PADDLE_THROW("Unsupported Variable Type of Ids");
    }

    size_t N = table_t->dims()[0];
    size_t D = table_t->dims()[1];
    auto* table = table_t->data<T>();
    auto* output = output_t->mutable_data<T>(context.GetPlace());

    dim3 threads(1, 1);
    dim3 grids(1, 1);

    if (padding_idx == -1)
      LookupTable<
          T, 1, 1, 1,
          false><<<grids, threads, 0, context.cuda_device_context().stream()>>>(
          output, table, ids, N, K, D, padding_idx);
    else
      LookupTable<
          T, 1, 1, 1,
          true><<<grids, threads, 0, context.cuda_device_context().stream()>>>(
          output, table, ids, N, K, D, padding_idx);

    framework::LoDTensor print_tensor_output;
    print_tensor_output.Resize(output_t->dims());
    // std::cout << print_tensor_output.dims() << std::endl;

    if (paddle::platform::is_cpu_place(output_t->place())) {
      print_tensor_table.ShareDataWith(*output_t);
    } else {
      // copy data to cpu to print
      paddle::platform::CPUPlace place;
      framework::TensorCopy(*output_t, place, &print_tensor_output);
    }
    Formater formater3;
    formater3.dtype = print_tensor_output.type();
    formater3.data = reinterpret_cast<void*>(print_tensor_output.data<void>());
    // formater3(print_tensor_output.numel());
  }
};

template <typename T>
class LookupTableGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    // std::cout << "Now we are in the backward kernel" << std::endl;
    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    bool is_sparse = context.Attr<bool>("is_sparse");
    // Since paddings are not trainable and fixed in forward, the gradient of
    // paddings makes no sense and we don't deal with it in backward.
    if (is_sparse) {
      // std::cout << "Is_sparse is true" << std::endl;
      auto* ids = context.Input<LoDTensor>("Ids");
      auto* table = context.Input<LoDTensor>("W");
      auto* d_output = context.Input<LoDTensor>(framework::GradVarName("Out"));
      auto* d_table = context.Output<SelectedRows>(framework::GradVarName("W"));

      auto* ids_data = ids->data<int64_t>();
      auto ids_dim = ids->dims();

      auto stream = dev_ctx.stream();
      // copy GPU memory to CPU pinned memory
      framework::Vector<int64_t> new_rows;
      new_rows.resize(ids_dim[0]);
      auto gpu_place = boost::get<platform::CUDAPlace>(context.GetPlace());

      // TODO(yuyang18): Strange code here.
      memory::Copy(platform::CPUPlace(),
                   new_rows.CUDAMutableData(context.GetPlace()), gpu_place,
                   ids_data, ids_dim[0] * sizeof(int64_t), stream);

      d_table->set_rows(new_rows);

      auto* d_table_value = d_table->mutable_value();
      d_table_value->Resize({ids_dim[0], table->dims()[1]});
      d_table_value->mutable_data<T>(context.GetPlace());

      auto* d_table_data = d_table_value->data<T>();
      auto* d_output_data = d_output->data<T>();
      PADDLE_ENFORCE_EQ(d_table_value->dims(), d_output->dims());
      memory::Copy(gpu_place, d_table_data, gpu_place, d_output_data,
                   d_output->numel() * sizeof(T), stream);

      framework::LoDTensor print_tensor_table;
      print_tensor_table.Resize(table->dims());
      // std::cout << "Printing table W" << std::endl;
      // std::cout << print_tensor_table.dims() << std::endl;

      if (paddle::platform::is_cpu_place(table->place())) {
        print_tensor_table.ShareDataWith(*table);
      } else {
        // copy data to cpu to print
        // std::cout << "Should be printed" << std::endl;
        paddle::platform::CPUPlace place;
        framework::TensorCopy(*table, place, &print_tensor_table);
      }
      Formater formater3;
      formater3.dtype = print_tensor_table.type();
      formater3.data = reinterpret_cast<void*>(print_tensor_table.data<void>());
      // formater3(print_tensor_table.numel());

      // Printing doutput
      framework::LoDTensor print_tensor_doutput;
      print_tensor_doutput.Resize(d_output->dims());
      // std::cout << "Printing d_output" << std::endl;
      // std::cout << print_tensor_doutput.dims() << std::endl;

      if (paddle::platform::is_cpu_place(d_output->place())) {
        print_tensor_doutput.ShareDataWith(*d_output);
      } else {
        // copy data to cpu to print
        paddle::platform::CPUPlace place;
        framework::TensorCopy(*d_output, place, &print_tensor_doutput);
      }
      Formater formater5;
      formater5.dtype = print_tensor_doutput.type();
      formater5.data =
          reinterpret_cast<void*>(print_tensor_doutput.data<void>());
      // formater5(print_tensor_doutput.numel());

    } else {
      // std::cout << "Is_sparse is false, hence we are here" << std::endl;
      auto ids_t = context.Input<LoDTensor>("Ids");
      auto d_output_t = context.Input<LoDTensor>(framework::GradVarName("Out"));
      auto d_table_t = context.Output<LoDTensor>(framework::GradVarName("W"));

      int N = d_table_t->dims()[0];
      int D = d_table_t->dims()[1];
      int K = ids_t->numel();
      const int64_t* ids = ids_t->data<int64_t>();
      const T* d_output = d_output_t->data<T>();
      T* d_table = d_table_t->mutable_data<T>(context.GetPlace());

      auto t = framework::EigenVector<T>::Flatten(*d_table_t);
      t.device(*dev_ctx.eigen_device()) = t.constant(static_cast<T>(0));

      dim3 threads(1, 1);
      dim3 grids(1, 1);
      LookupTableGrad<T, 1, 1, 1><<<grids, threads, 0, dev_ctx.stream()>>>(
          d_table, d_output, ids, N, K, D);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(lookup_table, ops::LookupTableCUDAKernel<float>,
                        ops::LookupTableCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(lookup_table_grad,
                        ops::LookupTableGradCUDAKernel<float>,
                        ops::LookupTableGradCUDAKernel<double>);
