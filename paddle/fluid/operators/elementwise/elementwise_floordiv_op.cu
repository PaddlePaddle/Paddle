#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_floordiv_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
struct CudaFloorDivFunctor {
  inline HOSTDEVICE T operator()(const T argv[]) const {
    PADDLE_ENFORCE(argv[1] != 0,
                   "InvalidArgument: divide by zero "
                   "encountered in floor-divide ops, please check.\n");
    return static_cast<T>(std::trunc(argv[0] / argv[1]));
  }
};

template <typename T>
class ElementwiseFloorDivKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    std::vector<const framework::Tensor*> ins;
    std::vector<framework::Tensor*> outs;
    const auto& cuda_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();

    int axis = PackTensorsIntoVector<T>(ctx, &ins, &outs);
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        cuda_ctx, ins, &outs, axis, CudaFloorDivFunctor<T>());
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_floordiv,
    ops::ElementwiseFloorDivKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseFloorDivKernel<plat::CUDADeviceContext, int64_t>);
