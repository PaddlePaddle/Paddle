/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_mul_op.h"
#include "paddle/phi/backends/gpu/gpu_context.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
class ElementwiseMulKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto x_var = ctx.InputVar("X");
    PADDLE_ENFORCE_EQ(x_var != nullptr, true,
                      platform::errors::InvalidArgument(
                          "Cannot get input Variable X, Variable name = %s.",
                          ctx.InputName("X")));
    const auto& cuda_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();
    if (x_var->IsType<phi::SelectedRows>()) {
      framework::Tensor x_for_selectedrows;
      std::vector<const framework::Tensor*> ins;
      std::vector<framework::Tensor*> outs;
      int axis =
          PackTensorsIntoVector<T>(ctx, &ins, &outs, &x_for_selectedrows);
      paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary,
                                                     T, T>(
          cuda_ctx, ins, &outs, axis, MulFunctor<T>());
    } else if (x_var->IsType<framework::LoDTensor>()) {
      auto* x_lod = ctx.Input<framework::LoDTensor>("X");
      auto* y_lod = ctx.Input<framework::LoDTensor>("Y");
      auto* z_lod = ctx.Output<framework::LoDTensor>("Out");
      z_lod->mutable_data<T>(ctx.GetPlace());

      int axis = ctx.Attr<int>("axis");
      auto pt_x = paddle::experimental::MakePhiDenseTensor(*x_lod);
      auto pt_y = paddle::experimental::MakePhiDenseTensor(*y_lod);
      auto pt_z = paddle::experimental::MakePhiDenseTensor(*z_lod);
      phi::MultiplyRawKernel<T>(static_cast<const phi::GPUContext&>(cuda_ctx),
                                *pt_x.get(), *pt_y.get(), axis, pt_z.get());
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "X's type[%s] is not supported by elementwise_op. X's type should be "
          "LoDTensor or SelectedRows.",
          framework::ToTypeName(x_var->Type())));
    }
  }
};

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, platform::CUDADeviceContext>::value>::type
ElementwiseMulGrad(const framework::ExecutionContext& ctx,
                   const framework::Tensor* x, const framework::Tensor* y,
                   const framework::Tensor* out, const framework::Tensor* dout,
                   framework::Tensor* dx, framework::Tensor* dy) {
  int axis = ctx.Attr<int>("axis");
  const auto& dev_ctx =
      ctx.template device_context<platform::CUDADeviceContext>();
  const auto place = ctx.GetPlace();

  if (dx != nullptr && dy != nullptr) {
    std::vector<const framework::Tensor*> ins = {dout, y, x};
    GetGradXAndYOut<ElementwiseType::kTernary, T>(
        dev_ctx, place, axis, ins, dout, dx, dy, MulGradXYFunctor<T, T>());
  } else if (dx != nullptr && dy == nullptr) {
    std::vector<const framework::Tensor*> ins = {dout, y};
    GetGradXOrYOut<ElementwiseType::kBinary, T>(dev_ctx, place, axis, ins, dout,
                                                dx, MulGradFunctor<T>());
  } else if (dx == nullptr && dy != nullptr) {
    std::vector<const framework::Tensor*> ins = {dout, x};
    GetGradXOrYOut<ElementwiseType::kBinary, T>(dev_ctx, place, axis, ins, dout,
                                                dy, MulGradFunctor<T>());
  }
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_mul, ops::ElementwiseMulKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, bool>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, plat::bfloat16>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, plat::complex<float>>,
    ops::ElementwiseMulKernel<plat::CUDADeviceContext, plat::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_mul_grad,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, bool>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext, plat::bfloat16>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext,
                                  plat::complex<float>>,
    ops::ElementwiseMulGradKernel<plat::CUDADeviceContext,
                                  plat::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_mul_grad_grad,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext, bool>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::bfloat16>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::complex<float>>,
    ops::ElementwiseMulDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_mul_triple_grad,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext, bool>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext,
                                        plat::bfloat16>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext,
                                        plat::complex<float>>,
    ops::ElementwiseMulTripleGradKernel<plat::CUDADeviceContext,
                                        plat::complex<double>>);
