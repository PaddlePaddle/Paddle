#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/elementwise/elementwise_sub_op.h"
#include "paddle/fluid/platform/complex.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
struct CudaSubFunctor {
  inline HOSTDEVICE T operator()(const T* args) const {
    return args[0] - args[1];
  }
};

template <typename T>
class ElementwiseSubKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    std::vector<const framework::Tensor*> ins;
    std::vector<framework::Tensor*> outs;
    const auto& cuda_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();

    int axis = PackTensorsIntoVector<T>(ctx, &ins, &outs);
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        cuda_ctx, ins, &outs, axis, CudaSubFunctor<T>());
  }
};

template <typename T>
static __global__ void SimpleElemwiseSubGradCUDAKernel(const T* dout,
                                                       int64_t size, T* dx,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    dx[col] = dout[col];
    dy[col] = -dout[col];
    col += blockDim.x * gridDim.x;
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
elementwise_sub_grad(const framework::ExecutionContext& ctx,
                     const framework::Tensor* x, const framework::Tensor* y,
                     const framework::Tensor* out,
                     const framework::Tensor* dout, framework::Tensor* dx,
                     framework::Tensor* dy) {
  dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
  auto size = x->numel();
  dim3 grid_size =
      dim3((size + PADDLE_CUDA_THREAD_SIZE - 1) / PADDLE_CUDA_THREAD_SIZE, 1);
  SimpleElemwiseSubGradCUDAKernel<
      T><<<grid_size, block_size, 0,
           ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
      dout->data<T>(), size, dx->mutable_data<T>(ctx.GetPlace()),
      dy->mutable_data<T>(ctx.GetPlace()));
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_sub,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<float>>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_sub_grad,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<float>>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_sub_grad_grad,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<float>>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<double>>);
