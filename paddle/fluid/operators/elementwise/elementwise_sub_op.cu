/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_sub_op.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    elementwise_sub,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_sub_grad,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  int64_t>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_sub_grad_grad,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>);
