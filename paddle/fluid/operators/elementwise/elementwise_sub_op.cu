#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/elementwise/elementwise_sub_op.h"
#include "paddle/fluid/operators/reduce_ops/reduce_functor_op.h"
#include "paddle/fluid/operators/reduce_ops/reduce_op.cu.h"
#include "paddle/fluid/platform/complex.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

//   template <typename T>
// static __global__ void SimpleElemwiseSubGradCUDAKernel(
//     const T* __restrict__ dout, int size, int vec_size, T* dx, T* dy) {
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
//   int stride = gridDim.x * blockDim.x;
//   int loop = size / vec_size;
//   int remainder = size % vec_size;
//   const float4* dout_vec = reinterpret_cast<const float4*>(dout);
//   float4* dx_vec = reinterpret_cast<float4*>(dx);
//   float4* dy_vec = reinterpret_cast<float4*>(dy);
//   float4 tmp_loop;

//   for (int i = tid; i < loop; i += stride) {
//     tmp_loop = dout_vec[i];
//     if(dx != nullptr){
//       dx_vec[i] = tmp_loop;
//     }
//     dy_vec[i] = -tmp_loop;
//   }

//   if (tid == loop && remainder != 0) {
//     T tmp_rem;
//     while (remainder) {
//       int idx = size - remainder;
//       remainder--;
//       tmp_rem = dout[idx];
//       if(dx != nullptr){
//         dx[idx] = tmp_rem;
//       }
//       dy[idx] = -tmp_rem;
//     }
//   }
// }

template <typename T>
static __global__ void SimpleElemwiseSubGradCUDAKernel(const T* dout,
                                                       int64_t size, T* dx,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    // if(dx != nullptr){
       dx[col] = dout[col];
    // }
    dy[col] = -dout[col];
    col += blockDim.x * gridDim.x;
  }
}

template <typename T>
static __global__ void SimpleElemwiseSubGradCUDAKernel(const T* dout,
                                                       int64_t size,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    dy[col] = -dout[col];
    col += blockDim.x * gridDim.x;
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, platform::CUDADeviceContext>::value>::type
default_elementwise_sub_grad(const framework::ExecutionContext& ctx,
                             const framework::Tensor* x,
                             const framework::Tensor* y,
                             const framework::Tensor* out,
                             const framework::Tensor* dout,
                             framework::Tensor* dx, framework::Tensor* dy) {
  int axis = ctx.Attr<int>("axis");
  auto* dout_data = dout->data<T>();
  // dx
  if (dx != nullptr) {
    auto* dx_data = dx->mutable_data<T>(ctx.GetPlace());
    if (dx->dims() == dout->dims()) {
      if (dx_data != dout_data) {
        framework::TensorCopy(
            *dout, ctx.GetPlace(),
            ctx.template device_context<platform::DeviceContext>(), dx);
      }
    } else {
      // For inplace strategy, dx will be stored in addr of dout, which makes
      // the result of dy wrong.
      if (dx->IsSharedBufferWith(*dout)) {
        dx->clear();
        dx->mutable_data<T>(x->dims(), ctx.GetPlace());
      }
      std::vector<int> reduce_dims = GetReduceDim(x->dims(), out->dims(), axis);
      gpuStream_t stream = ctx.cuda_device_context().stream();
      TensorReduceFunctorImpl<T, T, CustomSum>(*dout, dx, reduce_dims, stream);
    }
  }
  // dy
  if (dy != nullptr) {
    auto* dy_data = dy->mutable_data<T>(ctx.GetPlace());
    if (dy->dims() == dout->dims()) {
      if (dy_data != dout_data) {
        dim3 block_size = dim3(ELEMENTWISE_BLOCK_SIZE, 1);
        auto size = dy->numel();
        dim3 grid_size =
            dim3((size + ELEMENTWISE_BLOCK_SIZE - 1) / ELEMENTWISE_BLOCK_SIZE, 1);
        SimpleElemwiseSubGradCUDAKernel<
            T><<<grid_size, block_size, 0,
                ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
            dout->data<T>(), size,
            dy->mutable_data<T>(ctx.GetPlace()));
      }
    } else {
      std::vector<int> reduce_dims = GetReduceDim(y->dims(), out->dims(), axis);
      gpuStream_t stream = ctx.cuda_device_context().stream();
      TensorReduceFunctorImpl<T, T, CustomSub>(*dout, dy, reduce_dims, stream);
    }
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
elementwise_sub_grad(const framework::ExecutionContext& ctx,
                     const framework::Tensor* x, const framework::Tensor* y,
                     const framework::Tensor* out,
                     const framework::Tensor* dout, framework::Tensor* dx,
                     framework::Tensor* dy) {
        dim3 block_size = dim3(ELEMENTWISE_BLOCK_SIZE, 1);
        auto size = x->numel();
        dim3 grid_size =
            dim3((size + ELEMENTWISE_BLOCK_SIZE - 1) / ELEMENTWISE_BLOCK_SIZE, 1);
        SimpleElemwiseSubGradCUDAKernel<
            T><<<grid_size, block_size, 0,
                ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
            dout->data<T>(), size, dx->mutable_data<T>(ctx.GetPlace()),
            dy->mutable_data<T>(ctx.GetPlace()));
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_sub,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<float>>,
    ops::ElementwiseSubKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_sub_grad,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<float>>,
    ops::ElementwiseSubGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_sub_grad_grad,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<float>>,
    ops::ElementwiseSubDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<double>>);
