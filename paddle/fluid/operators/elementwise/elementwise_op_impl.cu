#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_op_impl.h"

namespace paddle {
namespace operators {

template <int Vec_size, typename T, typename Functor>
__device__ void VectorizedKernelHelper(SameDimsData<T> data, int size,
                                       Functor func, int tid) {
  using VecType = AlignedVector<T, Vec_size>;
  const VecType *x = reinterpret_cast<const VecType *>(data.in0);
  const VecType *y = reinterpret_cast<const VecType *>(data.in1);
  VecType *z = reinterpret_cast<VecType *>(data.out);
  VecType x_vec, y_vec, z_vec;
  x_vec = x_vec[tid];
  y_vec = y_vec[tid];
  T *x_slr = reinterpret_cast<T *>(&x_vec);
  T *y_slr = reinterpret_cast<T *>(&y_vec);
  T *z_slr = reinterpret_cast<T *>(&z_vec);

#pragma unroll
  for (int i = 0; i < Vec_size; ++i) {
    z_slr[i] = x_slr[i] + y_slr[i];
  }

  z[tid] = z_vec;
}

template <int Vec_size, typename T, typename Functor>
__global__ void VectorizedSameDimsKernel(SameDimsData<T> data, int size,
                                         Functor func) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int remain = size - Vec_size * tid;
  if (remain >= Vec_size) {
    VectorizedKernelHelper<Vec_size>(data, size, func, tid);
  } else {
    ScalarKernelHelper(data, size, func, tid * Vec_size, remain);
  }
}

template <typename T, typename Functor>
__device__ void ScalarKernelHelper(SameDimsData<T> data, int size, Functor func,
                                   int start, int remain) {
  for (int i = 0; i < remain; ++i) {
    T x = (data.in0)[start + i];
    T y = (data.in1)[start + i];
    (data.out)[start + i] = x + y;
  }
}

template <typename T, typename Functor>
__global__ void ScalarSameDimsKernel(SameDimsData<T> data, int size,
                                     Functor func) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  ScalarKernelHelper(data, size, func, tid, 0);
}

template <typename T, typename Functor>
void same_dims_launch_kernel(const framework::ExecutionContext &ctx,
                             SameDimsData<T> data, int64_t size, Functor func) {
  // calculate the max vec_size for all inputs and outputs
  int vec_size = data.GetVectorizedSize();
  int block_size = PADDLE_CUDA_THREAD_SIZE;
  int grid_size =
      ((size + vec_size - 1) / vec_size + block_size - 1) / block_size;
  // cuda kernel
  auto stream =
      ctx.template device_context<platform::CUDADeviceContext>().stream();
  switch (vec_size) {
    case 8:
      VectorizedSameDimsKernel<8><<<grid_size, block_size, 0, stream>>>(
          data, size, func);
      break;
    case 4:
      VectorizedSameDimsKernel<4><<<grid_size, block_size, 0, stream>>>(
          data, size, func);
      break;
    case 2:
      VectorizedSameDimsKernel<2><<<grid_size, block_size, 0, stream>>>(
          data, size, func);
      break;
    case 1:
      ScalarSameDimsKernel<<<grid_size, block_size, 0, stream>>>(data, size,
                                                                 func);
      break;
    default:
      VLOG(3) << "Unsupported vectorized size!";
  }
}

}  // namespace operators
}  // namespace paddle
