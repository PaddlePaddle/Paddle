#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_add_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_impl.cu.h"
#include "paddle/fluid/platform/complex128.h"
#include "paddle/fluid/platform/complex64.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

/*
   input: an array;
   return: the result of the math functor
   1. For Unary Op, the length of input array is 1,
      e.g. Relu: return args[0] > 0 ? args[0] : 0;
   2. For Binary Op, the length of input array is 2,
      e.g. Add: return args[0] + args[1];
*/
template <typename T>
struct CudaAddFunctor {
  __device__ __forceinline__ T operator()(const T* args) const {
    return args[0] + args[1];
  }
};

template <typename T>
struct SameDimsElemwiseAdd<platform::CUDADeviceContext, T> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor* x, const framework::Tensor* y,
                  framework::Tensor* z) {
    std::vector<const framework::Tensor*> ins = {x, y};
    std::vector<framework::Tensor*> outs = {z};
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        ctx.template device_context<platform::CUDADeviceContext>(), ins, &outs,
        CudaAddFunctor<T>());
  }
};

template <typename T>
static __global__ void SimpleElemwiseAddGradCUDAKernel(
    const T* __restrict__ dout, int size, int vec_size, T* dx, T* dy) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  int loop = size / vec_size;
  int remainder = size % vec_size;
  const float4* dout_vec = reinterpret_cast<const float4*>(dout);
  float4* dx_vec = reinterpret_cast<float4*>(dx);
  float4* dy_vec = reinterpret_cast<float4*>(dy);
  float4 tmp_loop;

  for (int i = tid; i < loop; i += stride) {
    tmp_loop = dout_vec[i];
    dx_vec[i] = tmp_loop;
    dy_vec[i] = tmp_loop;
  }

  if (tid == loop && remainder != 0) {
    T tmp_rem;
    while (remainder) {
      int idx = size - remainder;
      remainder--;
      tmp_rem = dout[idx];
      dx[idx] = tmp_rem;
      dy[idx] = tmp_rem;
    }
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
elementwise_add_grad(const framework::ExecutionContext& ctx,
                     const framework::Tensor* x, const framework::Tensor* y,
                     const framework::Tensor* out,
                     const framework::Tensor* dout, framework::Tensor* dx,
                     framework::Tensor* dy) {
  auto* dx_data = dx->mutable_data<T>(ctx.GetPlace());
  auto* dy_data = dy->mutable_data<T>(ctx.GetPlace());
  auto* dout_data = dout->data<T>();
  if (dx_data == dout_data && dy_data != dout_data) {
    VLOG(4) << "Special case when dx_data is the same as dout_data, "
               "only need copy dout to dy";
    framework::TensorCopy(
        *dout, ctx.GetPlace(),
        ctx.template device_context<platform::DeviceContext>(), dy);
  } else if (dx_data != dout_data && dy_data == dout_data) {
    VLOG(4) << "Special case when dy_data is the same as dout_data, "
               "only need copy dout to dx";
    framework::TensorCopy(
        *dout, ctx.GetPlace(),
        ctx.template device_context<platform::DeviceContext>(), dx);
  } else if (dx_data != dout_data && dy_data != dout_data) {
    auto size = x->numel();
    int vec_size = max(static_cast<int>(sizeof(float4) / sizeof(T)), 1);
    dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
    dim3 grid_size =
        dim3(((size + vec_size - 1) / vec_size + PADDLE_CUDA_THREAD_SIZE - 1) /
                 PADDLE_CUDA_THREAD_SIZE,
             1);
    SimpleElemwiseAddGradCUDAKernel<
        T><<<grid_size, block_size, 0,
             ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
        dout->data<T>(), size, vec_size, dx->mutable_data<T>(ctx.GetPlace()),
        dy->mutable_data<T>(ctx.GetPlace()));
  } else {
    VLOG(4) << "Special case when dy_data is the same as dout_data, "
               "and dx_data is the same as dout_data, do not need "
               "any operator";
  }
}

}  // namespace operators
}  // namespace paddle
REGISTER_OP_CUDA_KERNEL(
    elementwise_add, ops::ElementwiseAddKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex64>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_add_grad,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, plat::complex64>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, plat::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_add_grad_grad,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::complex64>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::complex128>);

REGISTER_OP_CUDA_KERNEL(
    grad_add, ops::ElementwiseAddKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex64>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex128>);
