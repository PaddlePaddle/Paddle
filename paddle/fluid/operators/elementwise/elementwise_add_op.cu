#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <algorithm>
#include <functional>
#include "paddle/fluid/operators/elementwise/elementwise_add_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_function.cu.h"
#include "paddle/fluid/platform/complex128.h"
#include "paddle/fluid/platform/complex64.h"
#include "paddle/fluid/platform/float16.h"

#define WARPSIZE 32

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
struct SameDimsElemwiseAdd<platform::CUDADeviceContext, T> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor* x, const framework::Tensor* y,
                  framework::Tensor* z) {
    AddRangeFunctor<T> functor(x->data<T>(), y->data<T>(), z->data<T>());
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx,
                                                              x->numel());
    for_range(functor);
  }
};

template <>
struct SameDimsElemwiseAdd<platform::CUDADeviceContext, platform::float16> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor* x, const framework::Tensor* y,
                  framework::Tensor* z) {
    auto size = x->numel();
    dim3 grid_size = dim3(((size + 1) / 2 + PADDLE_CUDA_THREAD_SIZE - 1) /
                              PADDLE_CUDA_THREAD_SIZE,
                          1);
    dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
    const half* x2 =
        reinterpret_cast<const half*>(x->data<platform::float16>());
    const half* y2 =
        reinterpret_cast<const half*>(y->data<platform::float16>());
    half* z2 = reinterpret_cast<half*>(z->data<platform::float16>());
    SameDimsElemwiseAddCUDAKernel<<<
        grid_size, block_size, 0,
        ctx.template device_context<platform::CUDADeviceContext>().stream()>>>(
        x2, y2, z2, size);
  }
};

template <typename T>
static __global__ void SimpleElemwiseAddGradCUDAKernel(const T* dout,
                                                       int64_t size, T* dx,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    dx[col] = dout[col];
    dy[col] = dout[col];
    col += blockDim.x * gridDim.x;
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
ElementwiseAddGrad(const framework::ExecutionContext& ctx,
                   const framework::Tensor* x, const framework::Tensor* y,
                   const framework::Tensor* out, const framework::Tensor* dout,
                   framework::Tensor* dx, framework::Tensor* dy) {
  dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
  auto size = x->numel();
  dim3 grid_size =
      dim3((size + PADDLE_CUDA_THREAD_SIZE - 1) / PADDLE_CUDA_THREAD_SIZE, 1);
  SimpleElemwiseAddGradCUDAKernel<
      T><<<grid_size, block_size, 0,
           ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
      dout->data<T>(), size, dx->mutable_data<T>(ctx.GetPlace()),
      dy->mutable_data<T>(ctx.GetPlace()));
}

inline static bool UseReduceFirstAxisRank1(const framework::DDim& dout_dims,
                                           const framework::DDim& x_dims,
                                           const framework::DDim& y_dims,
                                           const int axis) {
  int start_axis =
      (axis == -1 ? std::abs(x_dims.size() - y_dims.size()) : axis);

  if (y_dims[y_dims.size() - 1] == 1) {
    return false;
  }

  if (y_dims.size() > 1) {
    for (int i = 0; i < y_dims.size() - 1; ++i) {
      if (y_dims[i] != 1) {
        return false;
      }
    }
    return true;
  } else if (start_axis == x_dims.size() - 1) {
    return true;
  }
  return false;
}

inline static bool UseReduceFirstAxisRank2(const framework::DDim& dout_dims,
                                           const framework::DDim& x_dims,
                                           const framework::DDim& y_dims,
                                           const int axis) {
  int start_axis =
      (axis == -1 ? std::abs(x_dims.size() - y_dims.size()) : axis);

  if (y_dims.size() < 2 ||
      x_dims[x_dims.size() - 2] != y_dims[y_dims.size() - 2] ||
      x_dims[x_dims.size() - 1] != y_dims[y_dims.size() - 1]) {
    return false;
  }

  if (start_axis == x_dims.size() - 2) {
    return true;
  } else if (start_axis == 0) {
    for (int i = 0; i < y_dims.size() - 2; ++i) {
      if (y_dims[i] != 1) {
        return false;
      }
    }
    return true;
  }
  return false;
}

inline static bool UseReduceSecondAxisRank2(const framework::DDim& dout_dims,
                                            const framework::DDim& x_dims,
                                            const framework::DDim& y_dims,
                                            const int axis, int* start,
                                            int* end) {
  if (x_dims.size() != y_dims.size() || y_dims.size() < 3) {
    return false;
  }

  auto y_dims_vec = framework::vectorize(y_dims);
  auto start_iter = std::find(y_dims_vec.begin(), y_dims_vec.end(), 1);
  auto end_iter = std::find(y_dims_vec.rbegin(), y_dims_vec.rend(), 1);
  if (start_iter == y_dims_vec.end() || start_iter == y_dims_vec.end() - 1) {
    return false;
  } else {
    *start = std::distance(y_dims_vec.begin(), start_iter);
    *end = y_dims_vec.size() - 1 - std::distance(y_dims_vec.rbegin(), end_iter);
    for (int i = *start; i <= *end; ++i) {
      if (y_dims[i] != 1) {
        return false;
      }
    }
    return true;
  }
}

template <typename T, typename OP>
__global__ __launch_bounds__(1024) void ReduceFirstAixsKernel(
    const T* in, T* out, const int64_t num_rows, const int64_t num_cols, OP op,
    T init) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  T sum = init;
  if (row < num_rows && col < num_cols) sum = in[row * num_cols + col];

  __shared__ __align__(
      alignof(T)) char partial_sums_raw[WARPSIZE * (WARPSIZE + 1) * sizeof(T)];
  T* partial_sums = reinterpret_cast<T*>(partial_sums_raw);

  row += gridDim.y * blockDim.y;

  if (col < num_cols) {
    for (; row < num_rows; row += gridDim.y * blockDim.y) {
      sum = op(sum, in[row * num_cols + col]);
    }
  }

  partial_sums[threadIdx.x * (WARPSIZE + 1) + threadIdx.y] = sum;

  __syncthreads();

  if (threadIdx.y == 0 && col < num_cols) {
    T s = partial_sums[threadIdx.x * (WARPSIZE + 1)];

    const int numRowsThisBlock = min(static_cast<int64_t>(blockDim.y),
                                     num_rows - blockIdx.y * blockDim.y);

    for (int row = 1; row < numRowsThisBlock; ++row) {
      T t = partial_sums[threadIdx.x * (WARPSIZE + 1) + row];
      s = op(s, t);
    }

    out[col * gridDim.y + blockIdx.y] = s;
  }
}

template <typename DeviceContext, typename T>
static void ElemwiseYGradRank1CUDA(const framework::ExecutionContext& ctx,
                                   const framework::Tensor& dout,
                                   const int rows, const int cols,
                                   framework::Tensor* dx,
                                   framework::Tensor* dy) {
  dim3 block_dim(WARPSIZE, std::min(rows, 1024 / WARPSIZE));
  dim3 grid_dim((cols + (WARPSIZE - 1)) / WARPSIZE, 1, 1);

  if (dx) {
    dx->mutable_data<T>(ctx.GetPlace());
    framework::TensorCopy(
        dout, ctx.GetPlace(),
        ctx.template device_context<platform::DeviceContext>(), dx);
  }
  if (dy) {
    dy->mutable_data<T>(ctx.GetPlace());
    const T* dout_data = dout.data<T>();
    T* dy_data = dy->data<T>();
    auto stream = ctx.template device_context<DeviceContext>().stream();
    ReduceFirstAixsKernel<<<grid_dim, block_dim, 0, stream>>>(
        dout_data, dy_data, rows, cols, AddFunctor<T>(), static_cast<T>(0));
  }
}

template <typename T, typename OP>
__global__ __launch_bounds__(1024) void ReduceFirstOrSecondAxisKernel(
    const T* in, T* out, const int num_planes, const int num_rows,
    const int num_cols, OP op, T init) {
  const int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const int elems_per_plane = num_rows * num_cols;

  const int plane = gid / num_cols;
  const int col = gid % num_cols;

  if (plane >= num_planes) return;

  if (num_rows == 1) {
    out[plane * elems_per_plane + col] = in[plane * elems_per_plane + col];
    return;
  }

  T sum = op(in[plane * elems_per_plane + col],
             in[plane * elems_per_plane + num_cols + col]);
  for (int row = 2; row < num_rows; ++row) {
    sum = op(sum, in[plane * elems_per_plane + row * num_cols + col]);
  }

  out[plane * num_cols + col] = sum;
}

template <typename DeviceContext, typename T>
static void ElemwiseYGradRank2CUDA(const framework::ExecutionContext& ctx,
                                   const framework::Tensor& dout,
                                   const int planes, const int rows,
                                   const int cols, framework::Tensor* dx,
                                   framework::Tensor* dy) {
  int num_threads = 128;
  int num_blocks = (rows + num_threads - 1) / num_threads;

  if (planes != 1) {
    num_blocks = (planes * cols + num_threads - 1) / num_threads;
  }

  if (dx) {
    dx->mutable_data<T>(ctx.GetPlace());
    framework::TensorCopy(
        dout, ctx.GetPlace(),
        ctx.template device_context<platform::DeviceContext>(), dx);
  }
  if (dy) {
    dy->mutable_data<T>(ctx.GetPlace());
    const T* dout_data = dout.data<T>();
    T* dy_data = dy->data<T>();
    auto stream = ctx.template device_context<DeviceContext>().stream();
    ReduceFirstOrSecondAxisKernel<<<num_blocks, num_threads, 0, stream>>>(
        dout_data, dy_data, planes, rows, cols, AddFunctor<T>(),
        static_cast<T>(0));
  }
}

template <typename DeviceContext, typename T>
static bool ElemwiseGradUseReduce(const framework::ExecutionContext& ctx,
                                  const int axis, const framework::DDim x_dims,
                                  const framework::DDim y_dims,
                                  const framework::Tensor& dout,
                                  framework::Tensor* dx,
                                  framework::Tensor* dy) {
  int start = 0;
  int end = 0;
  auto x_dims_vec = framework::vectorize(x_dims);
  if (UseReduceFirstAxisRank1(dout.dims(), x_dims, y_dims, axis)) {
    int rows = std::accumulate(x_dims_vec.begin(), x_dims_vec.end() - 1, 1,
                               std::multiplies<int>());
    int cols = dx->dims()[dx->dims().size() - 1];
    if (cols > 512 && cols < 4096) {
      ElemwiseYGradRank1CUDA<DeviceContext, T>(ctx, dout, rows, cols, dx, dy);
      return true;
    }
  }

  if (UseReduceFirstAxisRank2(dout.dims(), x_dims, y_dims, axis)) {
    int rows = std::accumulate(x_dims_vec.begin(), x_dims_vec.end() - 2, 1,
                               std::multiplies<int>());
    int cols =
        dx->dims()[dx->dims().size() - 1] * dx->dims()[dx->dims().size() - 2];
    if (cols > 4096) {
      ElemwiseYGradRank2CUDA<DeviceContext, T>(ctx, dout, 1, rows, cols, dx,
                                               dy);
      return true;
    }
  }

  if (UseReduceSecondAxisRank2(dout.dims(), x_dims, y_dims, axis, &start,
                               &end)) {
    int planes = std::accumulate(x_dims_vec.begin(), x_dims_vec.begin() + start,
                                 1, std::multiplies<int>());
    int rows = std::accumulate(x_dims_vec.begin() + start,
                               x_dims_vec.begin() + end + 1, 1,
                               std::multiplies<int>());
    int cols = std::accumulate(x_dims_vec.begin() + end + 1, x_dims_vec.end(),
                               1, std::multiplies<int>());
    if (rows / (planes * cols) < 16) {
      ElemwiseYGradRank2CUDA<DeviceContext, T>(ctx, dout, planes, rows, cols,
                                               dx, dy);
      return true;
    }
  }

  return false;
}

template <typename T>
class ElementwiseAddGradKernel<platform::CUDADeviceContext, T>
    : public ElemwiseGradKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    ElemwiseGradKernel<T>::Compute(ctx);

    using Tensor = framework::Tensor;

    auto* x = ctx.Input<Tensor>("X");
    auto* y = ctx.Input<Tensor>("Y");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = ctx.Output<Tensor>(framework::GradVarName("Y"));
    // skip out
    auto* out = dout;
    int axis = ctx.Attr<int>("axis");

    // Special case when dy is not needed and dx doesn't reduce
    if (dx != nullptr && dy == nullptr && dx->dims() == dout->dims()) {
      VLOG(4) << "Special case when dy is not needed and dx doesn't "
                 "reduce";
      framework::TensorCopy(
          *dout, ctx.GetPlace(),
          ctx.template device_context<platform::DeviceContext>(), dx);
    } else if (dx == nullptr && dy != nullptr && dy->dims() == dout->dims()) {
      VLOG(4) << "Special case when dx is not needed and dy doesn't "
                 "reduce";
      framework::TensorCopy(
          *dout, ctx.GetPlace(),
          ctx.template device_context<platform::DeviceContext>(), dy);
    } else if (dx && dy && (dx->dims() == dy->dims())) {
      ElementwiseAddGrad<platform::CUDADeviceContext, T>(ctx, x, y, out, dout,
                                                         dx, dy);
    } else if (dx && dx->dims() == dout->dims() &&
               ElemwiseGradUseReduce<platform::CUDADeviceContext, T>(
                   ctx, axis, x->dims(), y->dims(), *dout, dx, dy)) {
    } else if (dy && dy->dims() == dout->dims() &&
               ElemwiseGradUseReduce<platform::CUDADeviceContext, T>(
                   ctx, axis, x->dims(), y->dims(), *dout, dy, dx)) {
    } else {
      DefaultElementwiseAddGrad<platform::CUDADeviceContext, T>(ctx, x, y, out,
                                                                dout, dx, dy);
    }
  }
};

}  // namespace operators
}  // namespace paddle
REGISTER_OP_CUDA_KERNEL(
    elementwise_add, ops::ElementwiseAddKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex64>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_add_grad,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, plat::complex64>,
    ops::ElementwiseAddGradKernel<plat::CUDADeviceContext, plat::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_add_grad_grad,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::complex64>,
    ops::ElementwiseAddDoubleGradKernel<plat::CUDADeviceContext,
                                        plat::complex128>);

REGISTER_OP_CUDA_KERNEL(
    grad_add, ops::ElementwiseAddKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, double>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::float16>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex64>,
    ops::ElementwiseAddKernel<plat::CUDADeviceContext, plat::complex128>);
