/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_div_op.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, platform::CUDADeviceContext>::value>::type
ElementwiseDivGrad(const framework::ExecutionContext& ctx,
                   const framework::Tensor* x, const framework::Tensor* y,
                   const framework::Tensor* out, const framework::Tensor* dout,
                   framework::Tensor* dx, framework::Tensor* dy) {
  int axis = ctx.Attr<int>("axis");
  const auto& dev_ctx = ctx.template device_context<DeviceContext>();
  const auto place = ctx.GetPlace();
  if (dx != nullptr && dy != nullptr) {
    std::vector<const framework::Tensor*> ins = {dout, out, y};
    GetGradXAndYOut<ElementwiseType::kTernary, T>(
        dev_ctx, place, axis, ins, dout, dx, dy, DivGradXYFunctor<T, T>());
  } else if (dx != nullptr && dy == nullptr) {
    std::vector<const framework::Tensor*> ins = {dout, y};
    GetGradXOrYOut<ElementwiseType::kBinary, T>(dev_ctx, place, axis, ins, dout,
                                                dx, DivGradXFunctor<T>());
  } else if (dy != nullptr && dx == nullptr) {
    std::vector<const framework::Tensor*> ins = {dout, out, y};
    GetGradXOrYOut<ElementwiseType::kTernary, T>(
        dev_ctx, place, axis, ins, dout, dy, DivGradYFunctor<T>());
  }
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_div,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::bfloat16>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<float>>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::bfloat16>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<float>>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad_grad,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::float16>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::bfloat16>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<float>>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<double>>);
