#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_div_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_function.cu.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_function.h"
#include "paddle/fluid/platform/complex128.h"
#include "paddle/fluid/platform/complex64.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
struct SameDimsElemwiseDiv<platform::CUDADeviceContext, T> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor* x, const framework::Tensor* y,
                  framework::Tensor* z) {
    DivRangeFunctor<T> functor(x->data<T>(), y->data<T>(), z->data<T>());
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx,
                                                              x->numel());
    for_range(functor);
  }
};

template <>
struct SameDimsElemwiseDiv<platform::CUDADeviceContext, platform::float16> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor* x, const framework::Tensor* y,
                  framework::Tensor* z) {
    auto size = x->numel();
    dim3 grid_size = dim3(((size + 1) / 2 + PADDLE_CUDA_THREAD_SIZE - 1) /
                              PADDLE_CUDA_THREAD_SIZE,
                          1);
    dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
    const half* x2 =
        reinterpret_cast<const half*>(x->data<platform::float16>());
    const half* y2 =
        reinterpret_cast<const half*>(y->data<platform::float16>());
    half* z2 = reinterpret_cast<half*>(z->data<platform::float16>());
    SameDimsElemwiseDivCUDAKernel<<<
        grid_size, block_size, 0,
        ctx.template device_context<platform::CUDADeviceContext>().stream()>>>(
        x2, y2, z2, size);
  }
};

template <typename T>
static __global__ void SimpleElemwiseDivGradCUDAKernel(const T* x, const T* y,
                                                       const T* out,
                                                       const T* dout,
                                                       int64_t size, T* dx,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    T o = dout[col];
    dx[col] = o / y[col];
    dy[col] = -o * out[col] / y[col];
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void SimpleElemwiseDivGradCUDAKernel<paddle::platform::complex64>(
    const paddle::platform::complex64* x, const paddle::platform::complex64* y,
    const paddle::platform::complex64* out,
    const paddle::platform::complex64* dout, int64_t size,
    paddle::platform::complex64* dx, paddle::platform::complex64* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    paddle::platform::complex64 o = dout[col];
    paddle::platform::complex64 y_conj(y[col].real, -y[col].imag);
    paddle::platform::complex64 out_div_y_conj((out[col] / y[col]).real,
                                               -(out[col] / y[col]).imag);
    dx[col] = o / y_conj;
    dy[col] = -o * out_div_y_conj;
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void SimpleElemwiseDivGradCUDAKernel<paddle::platform::complex128>(
    const paddle::platform::complex128* x,
    const paddle::platform::complex128* y,
    const paddle::platform::complex128* out,
    const paddle::platform::complex128* dout, int64_t size,
    paddle::platform::complex128* dx, paddle::platform::complex128* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    paddle::platform::complex128 o = dout[col];
    paddle::platform::complex128 y_conj(y[col].real, -y[col].imag);
    paddle::platform::complex128 out_div_y_conj((out[col] / y[col]).real,
                                                -(out[col] / y[col]).imag);
    dx[col] = o / y_conj;
    dy[col] = -o * out_div_y_conj;
    col += blockDim.x * gridDim.x;
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
elementwise_div_grad(const framework::ExecutionContext& ctx,
                     const framework::Tensor* x, const framework::Tensor* y,
                     const framework::Tensor* out,
                     const framework::Tensor* dout, framework::Tensor* dx,
                     framework::Tensor* dy) {
  dim3 block_size = dim3(PADDLE_CUDA_THREAD_SIZE, 1);
  auto size = x->numel();
  dim3 grid_size =
      dim3((size + PADDLE_CUDA_THREAD_SIZE - 1) / PADDLE_CUDA_THREAD_SIZE, 1);
  SimpleElemwiseDivGradCUDAKernel<
      T><<<grid_size, block_size, 0,
           ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
      x->data<T>(), y->data<T>(), out->data<T>(), dout->data<T>(), size,
      dx->mutable_data<T>(ctx.GetPlace()), dy->mutable_data<T>(ctx.GetPlace()));
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_div,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex64>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex64>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex128>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad_grad,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::float16>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex64>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex128>);
