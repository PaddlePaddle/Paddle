#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_div_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/reduce_ops/reduce_functor_op.h"
#include "paddle/fluid/operators/reduce_ops/reduce_op.cu.h"
#include "paddle/fluid/platform/complex.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T>
static __global__ void GARDYCUDAKernel(const T* y, int64_t size, T* t_y) {}

template <>
__global__ void GARDYCUDAKernel<paddle::platform::complex<float>>(
    const paddle::platform::complex<float>* y, int64_t size,
    paddle::platform::complex<float>* t_y) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    t_y[col].real = y[col].real;
    t_y[col].imag = -y[col].imag;
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void GARDYCUDAKernel<paddle::platform::complex<double>>(
    const paddle::platform::complex<double>* y, int64_t size,
    paddle::platform::complex<double>* t_y) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    t_y[col].real = y[col].real;
    t_y[col].imag = -y[col].imag;
    col += blockDim.x * gridDim.x;
  }
}

template <typename T>
static __global__ void SimpleElemwiseDivGradCUDAKernel(const T* x, const T* y,
                                                       const T* out,
                                                       const T* dout,
                                                       int64_t size, T* dx,
                                                       T* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    T o = dout[col];
    if (dx != nullptr) {
      dx[col] = o / y[col];
    }
    dy[col] = -o * out[col] / y[col];
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void
SimpleElemwiseDivGradCUDAKernel<paddle::platform::complex<float>>(
    const paddle::platform::complex<float>* x,
    const paddle::platform::complex<float>* y,
    const paddle::platform::complex<float>* out,
    const paddle::platform::complex<float>* dout, int64_t size,
    paddle::platform::complex<float>* dx,
    paddle::platform::complex<float>* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    paddle::platform::complex<float> o = dout[col];
    paddle::platform::complex<float> y_conj(y[col].real, -y[col].imag);
    paddle::platform::complex<float> out_div_y_conj((out[col] / y[col]).real,
                                                    -(out[col] / y[col]).imag);
    if (dx != nullptr) {
      dx[col] = o / y_conj;
    }
    dy[col] = -o * out_div_y_conj;
    col += blockDim.x * gridDim.x;
  }
}

template <>
__global__ void
SimpleElemwiseDivGradCUDAKernel<paddle::platform::complex<double>>(
    const paddle::platform::complex<double>* x,
    const paddle::platform::complex<double>* y,
    const paddle::platform::complex<double>* out,
    const paddle::platform::complex<double>* dout, int64_t size,
    paddle::platform::complex<double>* dx,
    paddle::platform::complex<double>* dy) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  while (col < size) {
    paddle::platform::complex<double> o = dout[col];
    paddle::platform::complex<double> y_conj(y[col].real, -y[col].imag);
    paddle::platform::complex<double> out_div_y_conj((out[col] / y[col]).real,
                                                     -(out[col] / y[col]).imag);
    if (dx != nullptr) {
      dx[col] = o / y_conj;
    }
    dy[col] = -o * out_div_y_conj;
    col += blockDim.x * gridDim.x;
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, platform::CUDADeviceContext>::value>::type
default_elementwise_div_grad(const framework::ExecutionContext& ctx,
                             const framework::Tensor* x,
                             const framework::Tensor* y,
                             const framework::Tensor* out,
                             const framework::Tensor* dout,
                             framework::Tensor* dx, framework::Tensor* dy) {
  int axis = ctx.Attr<int>("axis");
  auto* dout_data = dout->data<T>();
  dim3 block_size = dim3(ELEMENTWISE_BLOCK_SIZE, 1);
  // dx
  if (dx != nullptr) {
    auto* dx_data = dx->mutable_data<T>(ctx.GetPlace());
    if (dx->dims() == dout->dims()) {
      if (framework::IsComplexType(y->type())) {
        int size = y->numel();
        framework::Tensor t_y;
        t_y.Resize(y->dims());
        dim3 grid_size = dim3(
            (size + ELEMENTWISE_BLOCK_SIZE - 1) / ELEMENTWISE_BLOCK_SIZE, 1);
        GARDYCUDAKernel<T><<<
            grid_size, block_size, 0,
            ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
            y->data<T>(), size, t_y.mutable_data<T>(ctx.GetPlace()));
        const framework::Tensor* const_t_y =
            const_cast<const framework::Tensor*>(&t_y);
        default_elementwise_div<DeviceContext, T>(ctx, dout, const_t_y,
                                                  dx);  //  dout/y
      } else {
        default_elementwise_div<DeviceContext, T>(ctx, dout, y, dx);  //  dout/y
      }
    } else {
      // For inplace strategy, dx will be stored in addr of dout, which makes
      // the result of dy wrong.
      if (dx->IsSharedBufferWith(*dout)) {
        dx->clear();
        dx->mutable_data<T>(x->dims(), ctx.GetPlace());
      }
      framework::Tensor t_dx;
      t_dx.Resize(dout->dims());
      default_elementwise_div<DeviceContext, T>(ctx, dout, y, &t_dx);  // dout/y

      std::vector<int> reduce_dims = GetReduceDim(x->dims(), out->dims(), axis);
      gpuStream_t stream = ctx.cuda_device_context().stream();
      TensorReduceFunctorImpl<T, T, CustomSum>(t_dx, dx, reduce_dims, stream);
    }
  }
  // dy
  if (dy != nullptr) {
    auto* dy_data = dy->mutable_data<T>(ctx.GetPlace());
    if (dy->dims() == dout->dims()) {
      if (dy_data != dout_data) {
        // - out / y
        auto size = dy->numel();
        dim3 grid_size = dim3(
            (size + ELEMENTWISE_BLOCK_SIZE - 1) / ELEMENTWISE_BLOCK_SIZE, 1);
        SimpleElemwiseDivGradCUDAKernel<T><<<
            grid_size, block_size, 0,
            ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
            x->data<T>(), y->data<T>(), out->data<T>(), dout->data<T>(), size,
            nullptr, dy->mutable_data<T>(ctx.GetPlace()));
      }
    } else {
      framework::Tensor t_dy;
      t_dy.Resize(out->dims());
      default_elementwise_mul<DeviceContext, T>(ctx, dout, out,
                                                &t_dy);  // t_dy=dout*out
      framework::Tensor t_dyy;
      t_dyy.Resize(out->dims());
      const framework::Tensor* const_t_dy =
          const_cast<const framework::Tensor*>(&t_dy);
      if (framework::IsComplexType(y->type())) {
        int size = y->numel();
        framework::Tensor t_y;
        t_y.Resize(y->dims());
<<<<<<< 41291834704179377ab90d036093cf824e728ba7
        // paddle::platform::complex<double> y_conj(y[col].real, -y[col].imag);
=======
>>>>>>> add broadcast_div_bw
        dim3 grid_size = dim3(
            (size + ELEMENTWISE_BLOCK_SIZE - 1) / ELEMENTWISE_BLOCK_SIZE, 1);
        GARDYCUDAKernel<T><<<
            grid_size, block_size, 0,
            ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
            y->data<T>(), size, t_y.mutable_data<T>(ctx.GetPlace()));
        const framework::Tensor* const_t_y =
            const_cast<const framework::Tensor*>(&t_y);
        default_elementwise_div<DeviceContext, T>(ctx, const_t_dy, const_t_y,
                                                  &t_dyy);  // t_dy/y
      } else {
        default_elementwise_div<DeviceContext, T>(ctx, const_t_dy, y,
                                                  &t_dyy);  // t_dy/y
      }
      std::vector<int> reduce_dims = GetReduceDim(y->dims(), out->dims(), axis);
      gpuStream_t stream = ctx.cuda_device_context().stream();
      TensorReduceFunctorImpl<T, T, CustomSub>(t_dyy, dy, reduce_dims, stream);
    }
  }
}

template <typename DeviceContext, typename T>
typename std::enable_if<
    std::is_same<DeviceContext, plat::CUDADeviceContext>::value>::type
elementwise_div_grad(const framework::ExecutionContext& ctx,
                     const framework::Tensor* x, const framework::Tensor* y,
                     const framework::Tensor* out,
                     const framework::Tensor* dout, framework::Tensor* dx,
                     framework::Tensor* dy) {
  dim3 block_size = dim3(ELEMENTWISE_BLOCK_SIZE, 1);
  auto size = x->numel();
  dim3 grid_size =
      dim3((size + ELEMENTWISE_BLOCK_SIZE - 1) / ELEMENTWISE_BLOCK_SIZE, 1);
  SimpleElemwiseDivGradCUDAKernel<
      T><<<grid_size, block_size, 0,
           ctx.template device_context<plat::CUDADeviceContext>().stream()>>>(
      x->data<T>(), y->data<T>(), out->data<T>(), dout->data<T>(), size,
      dx->mutable_data<T>(ctx.GetPlace()), dy->mutable_data<T>(ctx.GetPlace()));
}

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_div,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::float16>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<float>>,
    ops::ElementwiseDivKernel<paddle::platform::CUDADeviceContext,
                              paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<float>>,
    ops::ElementwiseDivGradKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    elementwise_div_grad_grad,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::float16>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<float>>,
    ops::ElementwiseDivDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::complex<double>>);
