#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/elementwise/elementwise_mod_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename T, typename Enable = void>
struct CudaModFunctor {
  inline HOSTDEVICE T operator()(const T* args) const {
    T res = args[0] % args[1];

    // Accoding to #PR26732: in dividen % divsor
    // remainder shall have the same sign as divsor.
    if ((res != 0) && ((args[1] ^ res) < 0)) res += args[1];
    return res;
  }
};

template <typename T>
struct CudaModFunctor<
    T, typename std::enable_if_t<std::is_floating_point<T>::value>> {
  inline HOSTDEVICE T operator()(const T* args) const {
    T res = fmod(args[0], args[1]);

    // Accoding to #PR26732: in dividen % divsor
    // remainder shall have the same sign as divsor.
    if ((res != 0) && ((res < 0) != (args[1] < 0))) res += args[1];
    return res;
  }
};

template <typename T>
class ElementwiseModKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    std::vector<const framework::Tensor*> ins;
    std::vector<framework::Tensor*> outs;
    const auto& cuda_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();
    int axis = PackTensorsIntoVector<T>(ctx, &ins, &outs);
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        cuda_ctx, ins, &outs, axis, CudaModFunctor<T>());
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    elementwise_mod, ops::ElementwiseModKernel<plat::CUDADeviceContext, int>,
    ops::ElementwiseModKernel<plat::CUDADeviceContext, int64_t>,
    ops::ElementwiseModKernel<plat::CUDADeviceContext, float>,
    ops::ElementwiseModKernel<plat::CUDADeviceContext, double>);
