// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/operators/reduce_ops/reduce_mean_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n) : n_inv((T)(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};

template <typename T>
class ReduceMeanKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    bool reduce_all = context.Attr<bool>("reduce_all");
    auto* input = context.Input<Tensor>("X");
    auto* output = context.Output<Tensor>("Out");

    auto dims = context.Attr<std::vector<int>>("dim");
    bool keep_dim = context.Attr<bool>("keep_dim");

    std::vector<int> reduce_dims;
    if (reduce_all) {
      reduce_dims.resize(input->dims().size());
      for (int i = 0; i < reduce_dims.size(); ++i) reduce_dims[i] = i;
    } else {
      for (auto e : dims) {
        reduce_dims.push_back(e >= 0 ? e : e + input->dims().size());
      }
    }

    int reduce_num = 1;
    for (int i = 0; i < reduce_dims.size(); ++i) {
      reduce_num *= input->dims()[reduce_dims[i]];
    }

    auto stream = context.cuda_device_context().stream();
#ifdef PADDLE_WITH_HIP
    TensorReduce<T, T, hipcub::Sum, DivideFunctor<T>>(
        *input, output, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        DivideFunctor<T>(reduce_num), stream);
#else
    TensorReduce<T, T, hipcub::Sum, DivideFunctor<T>>(
        *input, output, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        DivideFunctor<T>(reduce_num), stream);
#endif
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(reduce_mean, ops::ReduceMeanKernel<float>,
                        ops::ReduceMeanKernel<double>);
