// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/operators/reduce_ops/frobenius_norm_op.h"

template <typename T>
using CUDAFrobeniusNormKernel =
    ops::ReduceKernel<paddle::platform::CUDADeviceContext, T,
                      ops::FrobeniusNormFunctor>;

REGISTER_OP_CUDA_KERNEL(frobenius_norm, CUDAFrobeniusNormKernel<float>,
                        CUDAFrobeniusNormKernel<double>);

template <typename T>
using CUDAFrobeniusNormGradKernel =
    ops::ReduceGradKernel<paddle::platform::CUDADeviceContext, T,
                          ops::FrobeniusNormGradFunctor>;

REGISTER_OP_CUDA_KERNEL(frobenius_norm_grad, CUDAFrobeniusNormGradKernel<float>,
                        CUDAFrobeniusNormGradKernel<double>);
