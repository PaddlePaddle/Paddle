// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/operators/reduce_ops/reduce_sum_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct IdentityFunctor {
  HOSTDEVICE explicit inline IdentityFunctor() {}

  HOSTDEVICE inline T operator()(const T& x) const { return x; }
};

template <typename T>
class ReduceSumKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    bool reduce_all = context.Attr<bool>("reduce_all");
    auto* input = context.Input<Tensor>("X");
    auto* output = context.Output<Tensor>("Out");
    auto out_dtype = context.Attr<int>("out_dtype");

    auto dims = context.Attr<std::vector<int>>("dim");
    bool keep_dim = context.Attr<bool>("keep_dim");

    std::vector<int> reduce_dims;
    if (reduce_all) {
      reduce_dims.resize(input->dims().size());
      for (int i = 0; i < reduce_dims.size(); ++i) reduce_dims[i] = i;
    } else {
      for (auto e : dims) {
        reduce_dims.push_back(e >= 0 ? e : e + input->dims().size());
      }
    }

    int reduce_num = 1;
    for (int i = 0; i < reduce_dims.size(); ++i) {
      reduce_num *= input->dims()[reduce_dims[i]];
    }

    auto stream = context.cuda_device_context().stream();
    if (out_dtype >= 0) {
      framework::VisitDataTypeSmall(
          static_cast<framework::proto::VarType::Type>(out_dtype),
          TensorReduceFunctor<T, hipcub::Sum, IdentityFunctor<T>>(
              *input, output, reduce_dims, static_cast<double>(0.0), hipcub::Sum(),
              IdentityFunctor<T>(), stream));
    } else {
      TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
          *input, output, reduce_dims, static_cast<T>(0), hipcub::Sum(),
          IdentityFunctor<T>(), stream);
    }
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(reduce_sum, ops::ReduceSumKernel<float>,
                        ops::ReduceSumKernel<double>, ops::ReduceSumKernel<int>,
                        ops::ReduceSumKernel<int64_t>);
