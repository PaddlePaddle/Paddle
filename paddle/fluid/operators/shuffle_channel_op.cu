#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/shuffle_channel_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__global__ void ShuffleChannel(const int nthreads, const int feature_map_size,
                               T* output, const T* input, int group_row,
                               int group_column, int len) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t ii = index; ii < nthreads; ii += offset) {
    const int n = index / group_row / group_column / len;
    const int i = (index / group_column / len) % group_row;
    const int j = index / len % group_column;
    const int k = index - (n * feature_map_size + (i * group_column + j) * len);
    T* p_o = output + n * feature_map_size + (j * group_row + i) * len;
    p_o[k] = input[index];
  }
}
template <typename DeviceContext, typename T>
class ShuffleChannelOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<framework::Tensor>("X");
    auto* output = ctx.Output<framework::Tensor>("Out");
    int group = ctx.Attr<int>("group");

    auto input_dims = input->dims();
    auto num = input_dims[0];
    auto channel = input_dims[1];
    auto height = input_dims[2];
    auto weight = input_dims[3];

    auto feature_map_size = channel * height * weight;
    auto sp_sz = height * weight;
    int group_row = group;
    int group_column = channel / group_row;
    // count is the product of NCHW same as numel()
    int count = num * group_column * group_row * sp_sz;

    int blocks = NumBlocks(output->numel());
    int threads = kNumCUDAThreads;

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());

    ShuffleChannel<
        T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        count, feature_map_size, output_data, input_data, group_row,
        group_column, sp_sz);
  }
};

template <typename DeviceContext, typename T>
class ShuffleChannelGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* output_grad =
        ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* input_grad =
        ctx.Output<framework::Tensor>(framework::GradVarName("X"));

    int group = ctx.Attr<int>("group");

    const auto& input_dims = input_grad->dims();
    auto num = input_dims[0];
    auto channel = input_dims[1];
    auto height = input_dims[2];
    auto weight = input_dims[3];
    auto feature_map_size = channel * height * weight;
    auto sp_sz = height * weight;

    int group_row = group;
    int group_column = channel / group_row;

    T* input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
    const T* output_grad_data = output_grad->data<T>();

    int blocks = NumBlocks(output_grad->numel());
    int threads = kNumCUDAThreads;
    int count = num * group_column * group_row * sp_sz;

    ShuffleChannel<
        T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        count, feature_map_size, input_grad_data, output_grad_data, group_row,
        group_column, sp_sz);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    shuffle_channel,
    ops::ShuffleChannelOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ShuffleChannelOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                    double>);
REGISTER_OP_CUDA_KERNEL(
    shuffle_channel_grad,
    ops::ShuffleChannelGradOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    ops::ShuffleChannelGradOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        double>);
