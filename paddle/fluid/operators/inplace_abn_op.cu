/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/inplace_abn_op.h"
#include "paddle/fluid/operators/sync_batch_norm_op.cu.h"
#include "paddle/pten/kernels/batch_norm_grad_kernel.h"
#include "paddle/pten/kernels/batch_norm_kernel.h"

namespace paddle {
namespace operators {

template <typename DeviceContext, typename T>
class InplaceABNKernel
    : public paddle::operators::SyncBatchNormKernel<DeviceContext, T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* y = ctx.Output<Tensor>("Y");
    auto* x = ctx.Input<Tensor>("X");
    PADDLE_ENFORCE_EQ(x, y, platform::errors::InvalidArgument(
                                "X and Y not inplaced in inplace mode"));
    auto activation =
        GetInplaceABNActivationType(ctx.Attr<std::string>("activation"));
    auto& place = *ctx.template device_context<DeviceContext>().eigen_device();

    if (ctx.Attr<bool>("use_sync_bn")) {
      SyncBatchNormKernel<DeviceContext, T>::Compute(ctx);
    } else {
      // BatchNormKernel<DeviceContext, T>::Compute(ctx);
      auto* scale = ctx.Input<Tensor>("Scale");
      auto* bias = ctx.Input<Tensor>("Bias");
      auto* mean = ctx.Input<Tensor>("Mean");
      auto* variance = ctx.Input<Tensor>("Variance");

      auto momentum = ctx.Attr<float>("momentum");
      auto epsilon = ctx.Attr<float>("epsilon");
      auto data_layout = ctx.Attr<std::string>("data_layout");
      auto is_test = ctx.Attr<bool>("is_test");
      auto use_global_stats = ctx.Attr<bool>("use_global_stats");
      auto trainable_statistics = ctx.Attr<bool>("trainable_statistics");
      auto fuse_with_relu = ctx.Attr<bool>("fuse_with_relu");

      auto* mean_out = ctx.Output<Tensor>("MeanOut");
      auto* variance_out = ctx.Output<Tensor>("VarianceOut");
      auto* saved_mean = ctx.Output<Tensor>("SavedMean");
      auto* saved_variance = ctx.Output<Tensor>("SavedVariance");
      auto* reserve_space = ctx.Output<Tensor>("ReserveSpace");

      auto& dev_ctx = ctx.device_context<DeviceContext>();
      pten::BatchNormKernel<T>(
          static_cast<const typename framework::ConvertToPtenContext<
              DeviceContext>::TYPE&>(dev_ctx),
          *x, *scale, *bias, *mean, *variance, momentum, epsilon, data_layout,
          is_test, use_global_stats, trainable_statistics, fuse_with_relu, y,
          mean_out, variance_out, saved_mean, saved_variance, reserve_space);
    }

    auto cur_y = EigenVector<T>::Flatten(*y);
    InplaceABNActivation<DeviceContext, T> functor;
    functor.Compute(ctx, activation, place, cur_y, cur_y);
  }
};

// Deriving the Gradient for the Backward Pass of Batch Normalization
// https://kevinzakka.github.io/2016/09/14/batch_normalization/
template <typename DeviceContext, typename T>
class InplaceABNGradKernel
    : public paddle::operators::SyncBatchNormGradKernel<DeviceContext, T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const auto* y = ctx.Input<Tensor>("Y");
    auto* d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto* d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    PADDLE_ENFORCE_EQ(d_x, d_y,
                      platform::errors::InvalidArgument(
                          "X@GRAD and Y@GRAD not inplaced in inplace mode"));
    auto& place = *ctx.template device_context<DeviceContext>().eigen_device();
    auto activation =
        GetInplaceABNActivationType(ctx.Attr<std::string>("activation"));

    auto py = *y;
    auto pd_y = *d_y;
    auto cur_y = EigenVector<T>::Flatten(py);
    auto cur_dy = EigenVector<T>::Flatten(pd_y);

    InplaceABNActivation<DeviceContext, T> functor;
    functor.GradCompute(ctx, activation, place, cur_y, cur_y, cur_dy, cur_dy);

    if (ctx.Attr<bool>("use_sync_bn")) {
      SyncBatchNormGradKernel<DeviceContext, T>::Compute(ctx);
    } else {
      auto* scale = ctx.Input<Tensor>("Scale");
      auto* bias = ctx.Input<Tensor>("Bias");
      auto* saved_mean = ctx.Input<Tensor>("SavedMean");
      auto* saved_variance = ctx.Input<Tensor>("SavedVariance");

      auto momentum = ctx.Attr<float>("momentum");
      auto epsilon = ctx.Attr<float>("epsilon");
      auto data_layout = ctx.Attr<std::string>("data_layout");
      auto is_test = ctx.Attr<bool>("is_test");
      auto use_global_stats = ctx.Attr<bool>("use_global_stats");
      auto trainable_statistics = ctx.Attr<bool>("trainable_statistics");
      auto fuse_with_relu = ctx.Attr<bool>("fuse_with_relu");

      auto* scale_grad = ctx.Output<Tensor>(framework::GradVarName("Scale"));
      auto* bias_grad = ctx.Output<Tensor>(framework::GradVarName("Bias"));

      auto* reserve_space = ctx.Input<Tensor>("ReserveSpace");
      auto* mean = ctx.Input<Tensor>("ReserveSpace");
      auto* variance = ctx.Input<Tensor>("ReserveSpace");

      paddle::optional<const Tensor&> space_opt = paddle::none;
      paddle::optional<const Tensor&> mean_opt = paddle::none;
      paddle::optional<const Tensor&> variance_opt = paddle::none;

      if (reserve_space != nullptr) {
        space_opt = *reserve_space;
      }

      if (mean != nullptr) {
        mean_opt = *mean;
      }

      if (variance != nullptr) {
        variance_opt = *variance;
      }

      auto& dev_ctx = ctx.device_context<DeviceContext>();
      pten::BatchNormGradRawKernel<T>(
          static_cast<const typename framework::ConvertToPtenContext<
              DeviceContext>::TYPE&>(dev_ctx),
          *d_y, *y, *scale, *bias, *saved_mean, *saved_variance, space_opt,
          mean_opt, variance_opt, momentum, epsilon, data_layout, is_test,
          use_global_stats, trainable_statistics, fuse_with_relu, true, d_x,
          scale_grad, bias_grad);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(inplace_abn,
                        ops::InplaceABNKernel<plat::CUDADeviceContext, float>);
REGISTER_OP_CUDA_KERNEL(
    inplace_abn_grad,
    ops::InplaceABNGradKernel<plat::CUDADeviceContext, float>);
#else
REGISTER_OP_CUDA_KERNEL(inplace_abn,
                        ops::InplaceABNKernel<plat::CUDADeviceContext, float>,
                        ops::InplaceABNKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    inplace_abn_grad, ops::InplaceABNGradKernel<plat::CUDADeviceContext, float>,
    ops::InplaceABNGradKernel<plat::CUDADeviceContext, double>);
#endif
