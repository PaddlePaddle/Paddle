#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/random_routing_op.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

#define CEIL(_x_, _y_) (((_x_)-1) / (_y_) + 1)
#define PERTHREAD_EXPERTS 256
#define WARP_SIZE 32

const int CUDA_NUM_THREADS = 512;
static inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

using LoDTensor = framework::LoDTensor;
using Tensor = framework::Tensor;

template <typename T>
__global__ void random_routing_kernel(int64_t* data,
                                      const int64_t length,
                                      const size_t N,
                                      const size_t D,
                                      const T* prob,
                                      const int64_t* topk_idx,
                                      const T* topk_value) {
  CUDA_KERNEL_LOOP(idx, length) {
    size_t row = idx / D;
    size_t col = idx % D;
    if (col != 1) return;
    if (static_cast<T>(2) * topk_value[idx] < prob[row]) {
      data[idx] = static_cast<int64_t>(-1);
    }
  }
}

template <typename T>
class RandomRoutingOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto topk_idx = context.Input<LoDTensor>("TopK_Idx");
    auto topk_value = context.Input<LoDTensor>("TopK_Value");
    auto prob = context.Input<LoDTensor>("Prob");
    auto out = context.Output<LoDTensor>("Out");

    auto place = context.GetPlace();
    const auto& dev_ctx = context.template device_context<phi::GPUContext>();
    framework::TensorCopy(*topk_idx, place, out);

    size_t N = topk_idx->dims()[0];
    size_t D = topk_idx->dims()[1];

    int64_t num_idx = topk_idx->numel();

    auto prob_data = prob->data<T>();
    auto topk_value_data = topk_value->data<T>();
    auto topk_idx_data = topk_idx->data<int64_t>();
    auto out_data = out->data<int64_t>();

    random_routing_kernel<T>
        <<<GET_BLOCKS(num_idx), CUDA_NUM_THREADS, 0, dev_ctx.stream()>>>(
            out_data, num_idx, N, D, prob_data, topk_idx_data, topk_value_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(random_routing,
                        ops::RandomRoutingOpCUDAKernel<float>,
                        ops::RandomRoutingOpCUDAKernel<double>,
                        ops::RandomRoutingOpCUDAKernel<plat::float16>);
