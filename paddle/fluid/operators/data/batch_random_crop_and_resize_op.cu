#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/data/batch_random_crop_and_resize_op.h"
#include "paddle/fluid/operators/reader/lod_tensor_blocking_queue.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {
namespace data {

using DataLayout = framework::DataLayout;
using LoDTensorBlockingQueueHolder =
    operators::reader::LoDTensorBlockingQueueHolder;

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const size_t idx_h, const size_t idx_w, const bool align_corners,
    const DataLayout data_format) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    // batch size
    int out_id_h = tid / output_w;
    // single image's index
    int out_id_w = tid % output_w;
    // input_w or output_w = c * h * w, img_size = h * w
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    // get output c, h, w index
    int channel_id, out_img_idy, out_img_idx;
    if (data_format == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    // get input h index with offset
    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy += idx_h;
    // get input w index with offset
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx += idx_w;

    if (data_format == DataLayout::kNCHW) {
      out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    } else {
      out[tid] = in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                    in_img_idx * num_channels + channel_id];
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const size_t idx_h, const size_t idx_w, const bool align_corners,
    const int align_mode, const DataLayout data_format) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    // batch size
    int out_id_h = tid / output_w;
    // single image's index
    int out_id_w = tid % output_w;
    // input_w or output_w = c * h * w, img_size = h * w
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    // get output c, h, w index
    int channel_id, out_img_idy, out_img_idx;
    if (data_format == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    // get input h index with offset
    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy + idx_h : idx_h;
    int h_id = (in_img_idy < in_img_h + idx_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h + idx_h : idx_h;
    T h1lambda = align_flag ? src_h - in_img_idy
                            : ratio_h * out_img_idy + idx_h - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    // get input w index with offset
    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx + idx_w : idx_w;
    int w_id = (in_img_idx < in_img_w + idx_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w + idx_w : idx_w;
    T w1lambda = align_flag ? src_w - in_img_idx
                            : ratio_w * out_img_idx + idx_w - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_format == DataLayout::kNCHW) {
      const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                            in_img_idy * in_img_w + in_img_idx];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                      w1lambda * in_pos[h_id * in_img_w + w_id]);
    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
              in_img_idx * num_channels + channel_id];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda *
              (w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w * num_channels] +
                      w1lambda * in_pos[h_id * in_img_w * num_channels +
                                        w_id * num_channels]);
    }
  }
}

template <typename T>
static void BatchRandomCropAndResizeFwd(
    const framework::ExecutionContext& ctx, const framework::Tensor& input,
    framework::Tensor* output, const std::vector<int64_t> out_size,
    const std::string interp_method, const bool align_corners,
    const int align_mode, const int img_h, const int img_w, const int c,
    const int idx_h, const int idx_w, const int crop_h, const int crop_w,
    const DataLayout data_format) {
  auto input_data = input.template data<T>();
  int out_h = static_cast<int>(out_size[0]);
  int out_w = static_cast<int>(out_size[1]);

  framework::DDim dim_out;
  if (data_format == DataLayout::kNCHW) {
    dim_out = {c, out_h, out_w};
  } else {
    dim_out = {out_h, out_w, c};
  }
  auto output_data = output->data<T>();

  if (img_h == crop_h && img_w == crop_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(crop_h - 1) / (out_h - 1)
                              : static_cast<float>(crop_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(crop_w - 1) / (out_w - 1)
                              : static_cast<float>(crop_w) / out_w;
  }

  int in_chw = c * crop_h * crop_w;
  int out_chw = c * out_h * out_w;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), out_chw);

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpFw<
        T><<<config.block_per_grid, config.thread_per_block, 0,
             ctx.cuda_device_context().stream()>>>(
        input_data, crop_h, crop_w, 1, in_chw, output_data, out_h, out_w, 1,
        out_chw, c, ratio_h, ratio_w, idx_h, idx_w, align_corners, data_format);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpFw<T><<<config.block_per_grid, config.thread_per_block, 0,
                            ctx.cuda_device_context().stream()>>>(
        input_data, crop_h, crop_w, 1, in_chw, output_data, out_h, out_w, 1,
        out_chw, c, ratio_h, ratio_w, idx_h, idx_w, align_corners, align_mode,
        data_format);
  }
}

template <typename T>
class BatchRandomCropAndResizeCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    // get input, output
    auto x = ctx.MultiInput<framework::Tensor>("X");
    PADDLE_ENFORCE_GT(x.size(), 0,
                      platform::errors::InvalidArgument(
                          "The size of X must be greater than 0."));
    auto* out = ctx.Output<framework::Tensor>("Out");

    auto aspect_ratio_min = ctx.Attr<float>("aspect_ratio_min");
    auto aspect_ratio_max = ctx.Attr<float>("aspect_ratio_max");
    AspectRatioRange aspect_ratio_range{aspect_ratio_min, aspect_ratio_max};

    auto area_min = ctx.Attr<float>("area_min");
    auto area_max = ctx.Attr<float>("area_max");
    AreaRange area_range{area_min, area_max};

    auto* generators = GeneratorManager::Instance()->GetGenerators(
        x.size(), x.size(), aspect_ratio_range, area_range);

    const std::vector<int64_t> size = ctx.Attr<std::vector<int64_t>>("size");

    // get data_format
    const std::string data_format_str = ctx.Attr<std::string>("data_format");
    const DataLayout data_format =
        framework::StringToDataLayout(data_format_str);
    // get interpolation method
    const std::string interp_method = ctx.Attr<std::string>("interp_method");
    bool align_corners = ctx.Attr<bool>("align_corners");
    int align_mode = ctx.Attr<int>("align_mode");

    auto* img = x.at(0);
    int64_t img_c =
        data_format == DataLayout::kNCHW ? img->dims()[0] : img->dims()[2];

    std::vector<int64_t> out_dim;
    if (data_format == DataLayout::kNCHW) {
      out_dim = {static_cast<int64_t>(x.size()), img_c, size[0], size[1]};
    } else {
      out_dim = {static_cast<int64_t>(x.size()), size[0], size[1], img_c};
    }
    out->Resize(phi::make_ddim(out_dim));
    out->mutable_data<T>(ctx.GetPlace());

    int img_h, img_w, idx_h, idx_w, crop_h, crop_w;
    for (int i = 0; i < x.size(); i++) {
      img = x.at(i);
      img_h =
          data_format == DataLayout::kNCHW ? img->dims()[1] : img->dims()[0];
      img_w =
          data_format == DataLayout::kNCHW ? img->dims()[2] : img->dims()[1];
      ROI roi;
      generators->at(i)->GenerateRandomROI(img_w, img_h, &roi);

      auto out_tensor = out->Slice(i, i + 1);
      BatchRandomCropAndResizeFwd<T>(ctx, *img, &out_tensor, size,
                                     interp_method, align_corners, align_mode,
                                     img_h, img_w, img_c, roi.y, roi.x, roi.h,
                                     roi.w, data_format);
    }
  }
};

}  // namespace data
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(batch_random_crop_and_resize,
                        ops::data::BatchRandomCropAndResizeCUDAKernel<uint8_t>,
                        ops::data::BatchRandomCropAndResizeCUDAKernel<float>,
                        ops::data::BatchRandomCropAndResizeCUDAKernel<double>);
