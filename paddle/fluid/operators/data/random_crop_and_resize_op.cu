#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/data/random_crop_and_resize_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/operators/reader/lod_tensor_blocking_queue.h"

namespace paddle {
namespace operators {
namespace data {

using framework::LoDTensor;
using DataLayout = framework::DataLayout;
using LoDTensorBlockingQueueHolder = operators::reader::LoDTensorBlockingQueueHolder;

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const size_t idx_h, const size_t idx_w, const bool align_corners,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    // batch size
    int out_id_h = tid / output_w;
    // single image's index
    int out_id_w = tid % output_w;
    // input_w or output_w = c * h * w, img_size = h * w
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    // get output c, h, w index
    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    // get input h index with offset
    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy += idx_h;
    // get input w index with offset
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx += idx_w;

    if (data_layout == DataLayout::kNCHW) {
      out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    } else {
      out[tid] = in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                    in_img_idx * num_channels + channel_id];
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const size_t idx_h, const size_t idx_w, const bool align_corners,
    const int align_mode, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    // batch size
    int out_id_h = tid / output_w;
    // single image's index
    int out_id_w = tid % output_w;
    // input_w or output_w = c * h * w, img_size = h * w
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    // get output c, h, w index
    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    // get input h index with offset
    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy + idx_h : idx_h;
    int h_id = (in_img_idy < in_img_h + idx_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h + idx_h : idx_h;
    T h1lambda = align_flag ? src_h - in_img_idy
                            : ratio_h * out_img_idy + idx_h - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    // get input w index with offset
    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx + idx_w : idx_w;
    int w_id = (in_img_idx < in_img_w + idx_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w + idx_w : idx_w;
    T w1lambda = align_flag ? src_w - in_img_idx
                            : ratio_w * out_img_idx + idx_w - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                            in_img_idy * in_img_w + in_img_idx];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                      w1lambda * in_pos[h_id * in_img_w + w_id]);
    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
              in_img_idx * num_channels + channel_id];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda *
              (w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w * num_channels] +
                      w1lambda * in_pos[h_id * in_img_w * num_channels +
                                        w_id * num_channels]);
    }
  }
}

template <typename T>
static void RandomCropAndResizeFwd(
    const framework::ExecutionContext& ctx, const framework::LoDTensor& input,
    framework::Tensor* output, const std::vector<int64_t> out_size,
    const std::string interp_method, const bool align_corners,
    const int align_mode, const int img_h, const int img_w, const int c,
    const int idx_h, const int idx_w, const int crop_h, const int crop_w,
    const DataLayout data_layout) {
  auto input_data = input.template data<T>();
  int out_h = static_cast<int>(out_size[0]);
  int out_w = static_cast<int>(out_size[1]);

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {c, out_h, out_w};
  } else {
    dim_out = {out_h, out_w, c};
  }
  auto output_data = output->data<T>();

  if (img_h == crop_h && img_w == crop_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(crop_h - 1) / (out_h - 1)
                              : static_cast<float>(crop_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(crop_w - 1) / (out_w - 1)
                              : static_cast<float>(crop_w) / out_w;
  }

  int in_chw = c * crop_h * crop_w;
  int out_chw = c * out_h * out_w;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), out_chw);

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpFw<
        T><<<config.block_per_grid, config.thread_per_block, 0,
             ctx.cuda_device_context().stream()>>>(
        input_data, crop_h, crop_w, 1, in_chw, output_data, out_h, out_w, 1,
        out_chw, c, ratio_h, ratio_w, idx_h, idx_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpFw<T><<<config.block_per_grid, config.thread_per_block, 0,
                            ctx.cuda_device_context().stream()>>>(
        input_data, crop_h, crop_w, 1, in_chw, output_data, out_h, out_w, 1,
        out_chw, c, ratio_h, ratio_w, idx_h, idx_w, align_corners, align_mode,
        data_layout);
  }
}

static void GetCropParameters(const int height, const int width,
                              const std::vector<float> scale,
                              const std::vector<float> ratio, int* idx_h,
                              int* idx_w, int* crop_h, int* crop_w,
                              const int seed, int num_attempts = 10) {
  double target_area, aspect_ratio;
  double area = height * width;
  std::vector<double> log_ratio;
  for (int i = 0; i < ratio.size(); i++)
    log_ratio.push_back(std::log(ratio[i]));
  std::default_random_engine engine(seed);
  std::uniform_real_distribution<double> dist_scale(scale[0], scale[1]);
  std::uniform_real_distribution<double> dist_log_ratio(log_ratio[0],
                                                        log_ratio[1]);

  for (int i = 0; i < num_attempts; i++) {
    target_area = dist_scale(engine) * area;
    aspect_ratio = std::exp(dist_log_ratio(engine));

    *crop_w =
        static_cast<int>(std::round(std::sqrt(target_area * aspect_ratio)));
    *crop_h =
        static_cast<int>(std::round(std::sqrt(target_area / aspect_ratio)));
    if (*crop_w > 0 && *crop_w <= width && *crop_h > 0 && *crop_h <= height) {
      std::uniform_int_distribution<int> dist_crop_h(0, height - *crop_h);
      *idx_h = dist_crop_h(engine);
      std::uniform_int_distribution<int> dist_crop_w(0, width - *crop_w);
      *idx_w = dist_crop_w(engine);
      return;
    }
  }

  // Fallback to central crop
  float in_ratio = static_cast<float>(width) / static_cast<float>(height);
  float min_ratio = ratio[0] > ratio[1] ? ratio[1] : ratio[0];
  float max_ratio = ratio[0] > ratio[1] ? ratio[0] : ratio[1];
  if (in_ratio < min_ratio) {
    *crop_w = width;
    *crop_h = static_cast<int>(std::round(*crop_w / min_ratio));
  } else if (in_ratio > max_ratio) {
    *crop_h = height;
    *crop_w = static_cast<int>(std::round(*crop_h * max_ratio));
  } else {
    // return whole image
    *crop_h = height;
    *crop_w = width;
  }
  *idx_h = (height - *crop_h) / 2;
  *idx_w = (width - *crop_w) / 2;
}

template <typename T>
class RandomCropAndResizeCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    // get input, output
    auto* x = ctx.Input<framework::LoDTensorArray>("X");
    PADDLE_ENFORCE_GT(x->size(), 0,
                      platform::errors::InvalidArgument(
                          "The size of X must be greater than 0."));
    auto* out = ctx.Output<framework::LoDTensor>("Out");

    // get size, scale, ratio
    auto size = ctx.Attr<std::vector<int64_t>>("size");
    auto scale = ctx.Attr<std::vector<float>>("scale");
    auto ratio = ctx.Attr<std::vector<float>>("ratio");
    // get random seed
    int seed = ctx.Attr<int>("seed");
    // get data_layout
    const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout data_layout =
        framework::StringToDataLayout(data_layout_str);
    // get interpolation method
    const std::string interp_method = ctx.Attr<std::string>("interp_method");
    bool align_corners = ctx.Attr<bool>("align_corners");
    int align_mode = ctx.Attr<int>("align_mode");

    auto* img = &x->at(0);
    int64_t img_c = data_layout == DataLayout::kNCHW ? \
                  img->dims()[0] : img->dims()[2];

    std::vector<int64_t> out_dim = {static_cast<int64_t>(x->size()),
                                    img_c, size[0], size[1]};
    out->Resize(phi::make_ddim(out_dim));
    out->mutable_data<T>(ctx.GetPlace());

    int img_h, img_w, idx_h, idx_w, crop_h, crop_w;
    for (int i = 0; i < x->size(); i++) {
      img = &x->at(i);
      img_h =
          data_layout == DataLayout::kNCHW ? img->dims()[1] : img->dims()[0];
      img_w =
          data_layout == DataLayout::kNCHW ? img->dims()[2] : img->dims()[1];
      GetCropParameters(img_h, img_w, scale, ratio, &idx_h, &idx_w, &crop_h,
                        &crop_w, seed);

      auto out_tensor = out->Slice(i, i + 1);
      RandomCropAndResizeFwd<T>(ctx, *img, &out_tensor, size, interp_method,
                                align_corners, align_mode, img_h, img_w, img_c,
                                idx_h, idx_w, crop_h, crop_w, data_layout);
    }
  }
};

}  // namespace data
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(random_crop_and_resize,
                        ops::data::RandomCropAndResizeCUDAKernel<uint8_t>,
                        ops::data::RandomCropAndResizeCUDAKernel<float>);
