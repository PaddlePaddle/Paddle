#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"

namespace paddle {
namespace operators {

template <typename T>
struct UniformGenerator {
  T min_, max_;
  unsigned int seed_;
  T diag_val_;
  unsigned int diag_num_;
  unsigned int diag_step_;
  __host__ __device__ UniformGenerator(T min, T max, int seed, int diag_num,
                                       int diag_step, T diag_val)
      : min_(min),
        max_(max),
        seed_(seed),
        diag_num_(diag_num),
        diag_step_(diag_step),
        diag_val_(diag_val) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n);
    T out = dist(rng);
    unsigned int remainder = n % (diag_step_ + 1);
    if (remainder == 0 && diag_num_ > n / (diag_step_ + 1)) {
      out = diag_val_;
    }
    return out;
  }
};

template <typename T>
struct UniformGeneratorOffset {
  T min_, max_;
  unsigned int seed_;
  T diag_val_;
  unsigned int diag_num_;
  unsigned int diag_step_;
  int offset_;
  __host__ __device__ UniformGeneratorOffset(T min, T max, int seed,
                                             int diag_num, int diag_step,
                                             T diag_val, int offset)
      : min_(min),
        max_(max),
        seed_(seed),
        diag_num_(diag_num),
        diag_step_(diag_step),
        diag_val_(diag_val),
        offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n + offset_);
    T out = dist(rng);
    unsigned int remainder = n % (diag_step_ + 1);
    if (remainder == 0 && diag_num_ > n / (diag_step_ + 1)) {
      out = diag_val_;
    }
    return out;
  }
};

template <typename T>
__global__ void fill_value(int64_t size, T* data, float value) {
  for (int idx = threadIdx.x; idx < size; idx += blockDim.x) {
    data[idx] = static_cast<T>(value);
  }
}

// It seems that Eigen::Tensor::random in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random as uniform_random_op.cu.
template <typename T>
class GPUUniformRandomInplaceKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto out_var = ctx.OutputVar("Out");
    auto* tensor = out_var->GetMutable<framework::LoDTensor>();
    T* data = tensor->mutable_data<T>(ctx.GetPlace());
    unsigned int seed = static_cast<unsigned int>(ctx.Attr<int>("seed"));
    bool seed_flag = false;
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
      seed_flag = true;
    }

    T min = static_cast<T>(ctx.Attr<float>("min"));
    T max = static_cast<T>(ctx.Attr<float>("max"));
    unsigned int diag_num =
        static_cast<unsigned int>(ctx.Attr<int>("diag_num"));
    unsigned int diag_step =
        static_cast<unsigned int>(ctx.Attr<int>("diag_step"));
    T diag_val = static_cast<T>(ctx.Attr<float>("diag_val"));
    thrust::counting_iterator<int64_t> index_sequence_begin(0);
    int64_t size = tensor->numel();
    int device_id =
        BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace()).GetDeviceId();
    auto gen_cuda = framework::GetDefaultCUDAGenerator(device_id);
    if (gen_cuda->GetIsInitPy() && seed_flag) {
      auto seed_offset = gen_cuda->IncrementOffset(1);
      int64_t gen_offset = size * seed_offset.second;
      thrust::transform(
          index_sequence_begin, index_sequence_begin + size,
          thrust::device_ptr<T>(data),
          UniformGeneratorOffset<T>(min, max, seed_offset.first, diag_num,
                                    diag_step, diag_val, gen_offset));
    } else {
      thrust::transform(
          index_sequence_begin, index_sequence_begin + size,
          thrust::device_ptr<T>(data),
          UniformGenerator<T>(min, max, seed, diag_num, diag_step, diag_val));
    }
  }
};

template <typename T>
class GPUUniformRandomInplaceGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
#ifdef __HIPCC__
    const int64_t kMaxBlockDim = 256;
#else
    const int64_t kMaxBlockDim = 512;
#endif
    auto* dx = ctx.Output<framework::Tensor>(framework::GradVarName("X"));
    auto* data = dx->mutable_data<T>(ctx.GetPlace());

    auto size = dx->numel();
    int64_t kBlockDim = std::min(size, kMaxBlockDim);
    fill_value<T><<<1, kBlockDim, 0>>>(size, data, static_cast<float>(0));
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    uniform_random_inplace,
    paddle::operators::GPUUniformRandomInplaceKernel<float>,
    paddle::operators::GPUUniformRandomInplaceKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    uniform_random_inplace_grad,
    paddle::operators::GPUUniformRandomInplaceGradKernel<float>,
    paddle::operators::GPUUniformRandomInplaceGradKernel<double>);
