#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// The file has been adapted from the two files:
//     https://github.com/laekov/fastmoe/blob/master/cuda/balancing.cu
//     https://github.com/laekov/fastmoe/blob/master/cuda/balancing.cuh
//     Git commit hash: 295a615aacce7e54a37e7935274ba15e901c78e4
// We retain the following license from the original files:
//     Copyright 2021, Jiaao He. All rights reserved.
//  Licensed under the Apache License, Version 2.0 (the "License").

#include "paddle/fluid/operators/prune_gate_by_capacity_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

DECLARE_bool(avoid_op_randomness);

namespace paddle {
namespace operators {
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T1, typename T2>
__global__ void prune_gate_by_capacity_kernel(const T1* gate_idx_data,
                                              T1* new_gate_idx_data,
                                              T2* expert_count_data,
                                              const int64_t batch_size) {
  CUDA_KERNEL_LOOP(i, batch_size) {
    auto orig_cap =
        platform::CudaAtomicAdd(expert_count_data + gate_idx_data[i], -1);
    if (orig_cap <= 0) {
      new_gate_idx_data[i] = -1;
    } else {
      new_gate_idx_data[i] = gate_idx_data[i];
    }
  }
}

template <typename DeviceContext, typename T1>
class PruneGateByCapacityFunctor {
 public:
  PruneGateByCapacityFunctor(const framework::ExecutionContext& context,
                             const framework::LoDTensor* gate_idx,
                             framework::LoDTensor* expert_count_out,
                             T1* new_gate_idx_data)
      : context_(context),
        gate_idx_(gate_idx),
        expert_count_out_(expert_count_out),
        new_gate_idx_data_(new_gate_idx_data) {}

  template <typename T2>
  void apply() {
    auto batch_size = gate_idx_->numel();
    auto* gate_idx_data = gate_idx_->data<T1>();

    auto& dev_ctx = context_.template device_context<DeviceContext>();
    auto* expert_count_out_data = expert_count_out_->data<T2>();

    int blocks = NumBlocks(batch_size);
    int threads = kNumCUDAThreads;

    prune_gate_by_capacity_kernel<T1, T2>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(gate_idx_data,
                                                   new_gate_idx_data_,
                                                   expert_count_out_data,
                                                   batch_size);
  }

 private:
  const framework::ExecutionContext context_;
  const framework::LoDTensor* gate_idx_;
  framework::LoDTensor* expert_count_out_;
  T1* new_gate_idx_data_;
};

template <typename Visitor>
static void VisitDataType(paddle::experimental::DataType type,
                          Visitor visitor) {
  if (type == paddle::experimental::DataType::INT64) {
    visitor.template apply<int64_t>();
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The received values gate_id type %s can not meet input requirements. "
        "Because the given gate_id data type of operators must be "
        "int64. Please input appropriate gate_id again! ",
        "framework::DataTypeToString(type)"));
  }
}

template <typename DeviceContext, typename T>
class PruneGateByCapacityCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* gate_idx = context.Input<LoDTensor>("GateIdx");
    auto* expert_count = context.Input<LoDTensor>("ExpertCount");
    // auto* expert_count_out = context.Output<LoDTensor>("ExpertCountOut");
    auto* new_gate_idx = context.Output<LoDTensor>("NewGateIdx");
    auto* new_gate_idx_data = new_gate_idx->mutable_data<T>(context.GetPlace());

    framework::LoDTensor expert_count_out;
    framework::TensorCopy(*expert_count, context.GetPlace(), &expert_count_out);
    PruneGateByCapacityFunctor<DeviceContext, T> functor(
        context, gate_idx, &expert_count_out, new_gate_idx_data);
    VisitDataType(expert_count->type(), functor);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    prune_gate_by_capacity,
    ops::PruneGateByCapacityCUDAKernel<phi::GPUContext, int64_t>);
