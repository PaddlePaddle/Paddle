#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/cast_op.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace operators {

template <typename InT, typename OutT>
__global__ void CastCUDAKernel(const InT* in, const int64_t N, OutT* out) {
  CUDA_KERNEL_LOOP(index, N) { out[index] = static_cast<OutT>(in[index]); }
}

template <typename InT>
struct CastOpFunctor<platform::CUDADeviceContext, InT> {
  const framework::Tensor* in_;
  framework::Tensor* out_;
  const platform::CUDADeviceContext& ctx_;
  CastOpFunctor(const framework::Tensor* in, framework::Tensor* out,
                const platform::CUDADeviceContext& ctx)
      : in_(in), out_(out), ctx_(ctx) {}

  template <typename OutT>
  void apply() const {
    auto* in = in_->data<InT>();
    auto size = in_->numel();
    auto* out = out_->mutable_data<OutT>(ctx_.GetPlace());
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(ctx_, size);
    CastCUDAKernel<InT, OutT><<<config.block_per_grid, config.thread_per_block,
                                0, ctx_.stream()>>>(in, size, out);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    cast, ops::CastOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, double>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, int>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, bool>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, uint8_t>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::float16>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::complex64>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::complex128>);
