#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/cast_op.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace operators {

// aligned vector generates vectorized load/store on CUDA
template <typename T, int Size>
struct alignas(sizeof(T) * Size) AlignedVector {
  T val[Size];
};

template <typename T>
inline int VectorizedSize(const T* pointer) {
  uint64_t address = reinterpret_cast<uint64_t>(pointer);
  constexpr int vec4 = std::alignment_of<AlignedVector<T, 4>>::value;  // NOLINT
  if (address % vec4 == 0) {
    return 4;
  }
  return 1;
}

template <typename InT, typename OutT, int VecSize>
__global__ void VecCastCUDAKernel(const InT* in, const int64_t N, OutT* out) {
  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  using LoadT = AlignedVector<InT, VecSize>;
  using StoreT = AlignedVector<OutT, VecSize>;
  for (int i = idx * VecSize; i < N; i += blockDim.x * gridDim.x * VecSize) {
    InT in_vec[VecSize];
    LoadT* in_value = reinterpret_cast<LoadT*>(&in_vec);
    *in_value = *reinterpret_cast<const LoadT*>(&in[i]);

    OutT out_vec[VecSize];
#pragma unroll
    for (int ii = 0; ii < VecSize; ii++) {
      out_vec[ii] = static_cast<OutT>(in_vec[ii]);
    }

    *(reinterpret_cast<StoreT*>(&out[i])) =
        *reinterpret_cast<StoreT*>(&out_vec[0]);
  }
}

template <typename InT, typename OutT>
__global__ void CastCUDAKernel(const InT* in, const int64_t N, OutT* out) {
  CUDA_KERNEL_LOOP(index, N) { out[index] = static_cast<OutT>(in[index]); }
}

template <typename InT>
struct CastOpFunctor<platform::CUDADeviceContext, InT> {
  const framework::Tensor* in_;
  framework::Tensor* out_;
  const platform::CUDADeviceContext& ctx_;
  CastOpFunctor(const framework::Tensor* in, framework::Tensor* out,
                const platform::CUDADeviceContext& ctx)
      : in_(in), out_(out), ctx_(ctx) {}

  template <typename OutT>
  void apply() const {
    auto* in = in_->data<InT>();
    auto size = in_->numel();
    auto* out = out_->mutable_data<OutT>(ctx_.GetPlace());
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(ctx_, size);
    int vec_size = VectorizedSize<OutT>(out);
    if (!std::is_same<InT, OutT>::value && vec_size == 4 && size % 4 == 0) {
      VecCastCUDAKernel<InT, OutT, 4><<<
          config.block_per_grid, config.thread_per_block, 0, ctx_.stream()>>>(
          in, size, out);
    } else {
      CastCUDAKernel<InT, OutT><<<config.block_per_grid,
                                  config.thread_per_block, 0, ctx_.stream()>>>(
          in, size, out);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    cast, ops::CastOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, double>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, int>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, bool>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext, uint8_t>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::float16>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::complex64>,
    ops::CastOpKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::complex128>);
