#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

using framework::Tensor;

template <typename T>
__global__ void Pad3DConstNCDHW(const int nthreads, const T* in_data,
                                const int num, const int channels,
                                const int in_depth, const int in_height,
                                const int in_width, const int out_depth,
                                const int out_height, const int out_width,
                                const int pad_front, const int pad_top,
                                const int pad_left, T value, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;
    out_data[index] =
        (in_d < 0 || in_h < 0 || in_w < 0 || in_d >= in_depth ||
         in_h >= in_height || in_w >= in_width)
            ? value
            : in_data[nc * in_depth * in_height * in_width +
                      in_d * in_height * in_width + in_h * in_width + in_w];
  }
}

template <typename T>
__global__ void Pad3DConstNDHWC(const int nthreads, const T* in_data,
                                const int num, const int channels,
                                const int in_depth, const int in_height,
                                const int in_width, const int out_depth,
                                const int out_height, const int out_width,
                                const int pad_front, const int pad_top,
                                const int pad_left, T value, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;
    const int in_d = out_d - pad_front;
    const int in_h = out_h - pad_top;
    const int in_w = out_w - pad_left;

    out_data[index] =
        (in_d < 0 || in_h < 0 || in_w < 0 || in_d >= in_depth ||
         in_h >= in_height || in_w >= in_width)
            ? value
            : in_data[n * in_depth * in_height * in_width * channels +
                      in_d * in_height * in_width * channels +
                      in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DReflectNCDHW(const int nthreads, const T* in_data,
                                  const int num, const int channels,
                                  const int in_depth, const int in_height,
                                  const int in_width, const int out_depth,
                                  const int out_height, const int out_width,
                                  const int pad_front, const int pad_top,
                                  const int pad_left, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);                     // reflect by 0
    in_d = min(in_d, 2 * in_depth - in_d - 2);   // reflect by in_depth
    in_h = max(in_h, -in_h);                     // reflect by 0
    in_h = min(in_h, 2 * in_height - in_h - 2);  // reflect by in_height
    in_w = max(in_w, -in_w);                     // reflect by 0
    in_w = min(in_w, 2 * in_width - in_w - 2);   // reflect by in_width
    out_data[index] =
        in_data[(nc * in_depth * in_height + in_d * in_height + in_h) *
                    in_width +
                in_w];
  }
}

template <typename T>
__global__ void Pad3DReflectNDHWC(const int nthreads, const T* in_data,
                                  const int num, const int channels,
                                  const int in_depth, const int in_height,
                                  const int in_width, const int out_depth,
                                  const int out_height, const int out_width,
                                  const int pad_front, const int pad_top,
                                  const int pad_left, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;
    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);
    in_d = min(in_d, 2 * in_depth - in_d - 2);
    in_h = max(in_h, -in_h);
    in_h = min(in_h, 2 * in_height - in_h - 2);
    in_w = max(in_w, -in_w);
    in_w = min(in_w, 2 * in_width - in_w - 2);

    out_data[index] = in_data[n * in_depth * in_height * in_width * channels +
                              in_d * in_height * in_width * channels +
                              in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DReplicateNCDHW(const int nthreads, const T* in_data,
                                    const int num, const int channels,
                                    const int in_depth, const int in_height,
                                    const int in_width, const int out_depth,
                                    const int out_height, const int out_width,
                                    const int pad_front, const int pad_top,
                                    const int pad_left, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    out_data[index] =
        in_data[(nc * in_depth * in_height + in_d * in_height + in_h) *
                    in_width +
                in_w];
  }
}

template <typename T>
__global__ void Pad3DReplicateNDHWC(const int nthreads, const T* in_data,
                                    const int num, const int channels,
                                    const int in_depth, const int in_height,
                                    const int in_width, const int out_depth,
                                    const int out_height, const int out_width,
                                    const int pad_front, const int pad_top,
                                    const int pad_left, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    out_data[index] = in_data[n * in_depth * in_height * in_width * channels +
                              in_d * in_height * in_width * channels +
                              in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DCircularNCDHW(const int nthreads, const T* in_data,
                                   const int num, const int channels,
                                   const int in_depth, const int in_height,
                                   const int in_width, const int out_depth,
                                   const int out_height, const int out_width,
                                   const int pad_front, const int pad_top,
                                   const int pad_left, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    out_data[index] =
        in_data[(nc * in_depth * in_height + in_d * in_height + in_h) *
                    in_width +
                in_w];
  }
}

template <typename T>
__global__ void Pad3DCircularNDHWC(const int nthreads, const T* in_data,
                                   const int num, const int channels,
                                   const int in_depth, const int in_height,
                                   const int in_width, const int out_depth,
                                   const int out_height, const int out_width,
                                   const int pad_front, const int pad_top,
                                   const int pad_left, T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    out_data[index] = in_data[n * in_depth * in_height * in_width * channels +
                              in_d * in_height * in_width * channels +
                              in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DGradConstNCDHW(const int in_size, T* d_in_data,
                                    const int num, const int channels,
                                    const int in_depth, const int in_height,
                                    const int in_width, const int out_depth,
                                    const int out_height, const int out_width,
                                    const int pad_front, const int pad_top,
                                    const int pad_left, const T* d_out_data) {
  CUDA_KERNEL_LOOP(in_index, in_size) {
    const int in_w = in_index % in_width;

    int nc = in_index / in_width;
    const int in_h = nc % in_height;

    nc /= in_height;
    const int in_d = nc % in_depth;

    nc /= in_depth;

    const int out_d = in_d + pad_front;
    const int out_h = in_h + pad_top;
    const int out_w = in_w + pad_left;
    d_in_data[in_index] =
        d_out_data[nc * out_depth * out_height * out_width +
                   out_d * out_height * out_width + out_h * out_width + out_w];
  }
}

template <typename T>
__global__ void Pad3DGradConstNDHWC(const int in_size, T* d_in_data,
                                    const int num, const int channels,
                                    const int in_depth, const int in_height,
                                    const int in_width, const int out_depth,
                                    const int out_height, const int out_width,
                                    const int pad_front, const int pad_top,
                                    const int pad_left, const T* d_out_data) {
  CUDA_KERNEL_LOOP(in_index, in_size) {
    const int c = in_index % channels;
    int n = in_index / channels;

    const int in_w = n % in_width;
    n /= in_width;

    const int in_h = n % in_height;
    n /= in_height;

    const int in_d = n % in_depth;
    n /= in_depth;

    const int out_d = in_d + pad_front;
    const int out_h = in_h + pad_top;
    const int out_w = in_w + pad_left;

    d_in_data[in_index] =
        d_out_data[n * out_depth * out_height * out_width * channels +
                   out_d * out_height * out_width * channels +
                   out_h * out_width * channels + out_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DGradReflectNCDHW(const int out_size, T* d_in_data,
                                      const int num, const int channels,
                                      const int in_depth, const int in_height,
                                      const int in_width, const int out_depth,
                                      const int out_height, const int out_width,
                                      const int pad_front, const int pad_top,
                                      const int pad_left, const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    int nc = out_index / out_width;
    const int out_w = out_index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);
    in_h = max(in_h, -in_h);
    in_w = max(in_w, -in_w);

    in_d = min(in_d, 2 * in_depth - in_d - 2);
    in_h = min(in_h, 2 * in_height - in_h - 2);
    in_w = min(in_w, 2 * in_width - in_w - 2);

    platform::CudaAtomicAdd(
        &d_in_data[nc * in_depth * in_height * in_width +
                   in_d * in_height * in_width + in_h * in_width + in_w],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradReflectNDHWC(const int out_size, T* d_in_data,
                                      const int num, const int channels,
                                      const int in_depth, const int in_height,
                                      const int in_width, const int out_depth,
                                      const int out_height, const int out_width,
                                      const int pad_front, const int pad_top,
                                      const int pad_left, const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    const int c = out_index % channels;
    int n = out_index / channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);
    in_h = max(in_h, -in_h);
    in_w = max(in_w, -in_w);

    in_d = min(in_d, in_depth * 2 - in_d - 2);
    in_h = min(in_h, in_height * 2 - in_h - 2);
    in_w = min(in_w, in_width * 2 - in_w - 2);
    platform::CudaAtomicAdd(
        &d_in_data[n * in_depth * in_height * in_width * channels +
                   in_d * in_height * in_width * channels +
                   in_h * in_width * channels + in_w * channels + c],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradReplicateNCDHW(
    const int out_size, T* d_in_data, const int num, const int channels,
    const int in_depth, const int in_height, const int in_width,
    const int out_depth, const int out_height, const int out_width,
    const int pad_front, const int pad_top, const int pad_left,
    const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    int nc = out_index / out_width;
    const int out_w = out_index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    const int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    const int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    const int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    platform::CudaAtomicAdd(
        &d_in_data[nc * in_depth * in_height * in_width +
                   in_d * in_height * in_width + in_h * in_width + in_w],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradReplicateNDHWC(
    const int out_size, T* d_in_data, const int num, const int channels,
    const int in_depth, const int in_height, const int in_width,
    const int out_depth, const int out_height, const int out_width,
    const int pad_front, const int pad_top, const int pad_left,
    const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    const int c = out_index % channels;
    int n = out_index / channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    const int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    const int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    const int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    platform::CudaAtomicAdd(
        &d_in_data[n * in_depth * in_height * in_width * channels +
                   in_d * in_height * in_width * channels +
                   in_h * in_width * channels + in_w * channels + c],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradCircularNCDHW(const int out_size, T* d_in_data,
                                       const int num, const int channels,
                                       const int in_depth, const int in_height,
                                       const int in_width, const int out_depth,
                                       const int out_height,
                                       const int out_width, const int pad_front,
                                       const int pad_top, const int pad_left,
                                       const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    int nc = out_index / out_width;
    const int out_w = out_index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    platform::CudaAtomicAdd(
        &d_in_data[nc * in_depth * in_height * in_width +
                   in_d * in_height * in_width + in_h * in_width + in_w],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradCircularNDHWC(const int out_size, T* d_in_data,
                                       const int num, const int channels,
                                       const int in_depth, const int in_height,
                                       const int in_width, const int out_depth,
                                       const int out_height,
                                       const int out_width, const int pad_front,
                                       const int pad_top, const int pad_left,
                                       const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    const int c = out_index % channels;
    int n = out_index / channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    platform::CudaAtomicAdd(
        &d_in_data[n * in_depth * in_height * in_width * channels +
                   in_d * in_height * in_width * channels +
                   in_h * in_width * channels + in_w * channels + c],
        d_out_data[out_index]);
  }
}

static inline std::vector<int> GetPaddings(
    const framework::ExecutionContext& context) {
  std::vector<int> paddings(6);
  auto* paddings_data = context.Input<Tensor>("Paddings");
  if (paddings_data) {
    Tensor pads;
    framework::TensorCopySync(*paddings_data, platform::CPUPlace(), &pads);
    auto pads_data = pads.data<int>();
    std::memcpy(paddings.data(), pads_data, paddings.size() * sizeof(int));
  } else {
    auto pads = context.Attr<std::vector<int>>("paddings");
    std::copy(pads.begin(), pads.end(), paddings.data());
  }
  return paddings;
}

template <typename T>
class Pad3dCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    std::vector<int> pads = GetPaddings(context);
    auto mode = context.Attr<std::string>("mode");
    auto data_format = context.Attr<std::string>("data_format");
    T value = static_cast<T>(context.Attr<float>("value"));

    auto* x = context.Input<Tensor>("X");
    auto in_dims = x->dims();
    const T* in_data = x->data<T>();
    auto* out = context.Output<Tensor>("Out");
    auto out_dims = out->dims();
    if (data_format == "NCDHW") {
      out_dims[0] = in_dims[0];
      out_dims[1] = in_dims[1];
      out_dims[2] = in_dims[2] + pads[4] + pads[5];
      out_dims[3] = in_dims[3] + pads[2] + pads[3];
      out_dims[4] = in_dims[4] + pads[0] + pads[1];
    } else {
      out_dims[0] = in_dims[0];
      out_dims[1] = in_dims[1] + pads[4] + pads[5];
      out_dims[2] = in_dims[2] + pads[2] + pads[3];
      out_dims[3] = in_dims[3] + pads[0] + pads[1];
      out_dims[4] = in_dims[4];
    }
    T* out_data = out->mutable_data<T>(out_dims, context.GetPlace());

    int channels = in_dims[1];
    int in_depth = in_dims[2];
    int in_height = in_dims[3];
    int in_width = in_dims[4];
    int out_depth = out_dims[2];
    int out_height = out_dims[3];
    int out_width = out_dims[4];
    if (data_format == "NDHWC") {
      channels = in_dims[4];
      in_depth = in_dims[1];
      in_height = in_dims[2];
      in_width = in_dims[3];
      out_depth = out_dims[1];
      out_height = out_dims[2];
      out_width = out_dims[3];
    }

    if (mode == "reflect") {
      PADDLE_ENFORCE_GT(in_depth, pads[4],
                        platform::errors::InvalidArgument(
                            "The depth of Input(X)'s dimension should be "
                            "greater than pad_front"
                            " in reflect mode"
                            ", but received depth(%d) and pad_front(%d).",
                            in_depth, pads[4]));
      PADDLE_ENFORCE_GT(in_depth, pads[5],
                        platform::errors::InvalidArgument(
                            "The depth of Input(X)'s dimension should be "
                            "greater than pad_back"
                            " in reflect mode"
                            ", but received depth(%d) and pad_back(%d).",
                            in_depth, pads[5]));

      PADDLE_ENFORCE_GT(in_height, pads[2],
                        platform::errors::InvalidArgument(
                            "The height of Input(X)'s dimension should be "
                            "greater than pad_top"
                            " in reflect mode"
                            ", but received depth(%d) and pad_top(%d).",
                            in_height, pads[2]));
      PADDLE_ENFORCE_GT(in_height, pads[3],
                        platform::errors::InvalidArgument(
                            "The height of Input(X)'s dimension should be "
                            "greater than pad_bottom"
                            " in reflect mode"
                            ", but received depth(%d) and pad_bottom(%d).",
                            in_height, pads[3]));

      PADDLE_ENFORCE_GT(in_width, pads[0],
                        platform::errors::InvalidArgument(
                            "The width of Input(X)'s dimension should be "
                            "greater than pad_left"
                            " in reflect mode"
                            ", but received depth(%d) and pad_left(%d).",
                            in_width, pads[0]));
      PADDLE_ENFORCE_GT(in_width, pads[1],
                        platform::errors::InvalidArgument(
                            "The width of Input(X)'s dimension should be "
                            "greater than pad_right"
                            " in reflect mode"
                            ", but received depth(%d) and pad_right(%d).",
                            in_width, pads[1]));
    }

    const int pad_left = pads[0];
    const int pad_top = pads[2];
    const int pad_front = pads[4];
    const int num = in_dims[0];

    auto stream = context.cuda_device_context().stream();
    int block = PADDLE_CUDA_NUM_THREADS;
    const int out_size = out->numel();
    int grid = (out_size + block - 1) / block;

    if (data_format == "NCDHW") {
      if (mode == "reflect") {
        Pad3DReflectNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            out_data);
      } else if (mode == "replicate") {
        Pad3DReplicateNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            out_data);
      } else if (mode == "circular") {
        Pad3DCircularNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            out_data);
      } else {
        Pad3DConstNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            value, out_data);
      }
    } else {
      if (mode == "reflect") {
        Pad3DReflectNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            out_data);
      } else if (mode == "replicate") {
        Pad3DReplicateNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            out_data);
      } else if (mode == "circular") {
        Pad3DCircularNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            out_data);
      } else {
        Pad3DConstNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            value, out_data);
      }
    }
  }
};

template <typename T>
class Pad3dGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    std::vector<int> pads = GetPaddings(context);
    auto mode = context.Attr<std::string>("mode");
    auto data_format = context.Attr<std::string>("data_format");
    auto* d_out = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* d_in = context.Output<Tensor>(framework::GradVarName("X"));
    auto d_in_dims = d_in->dims();
    auto d_out_dims = d_out->dims();
    const T* d_out_data = d_out->data<T>();
    T* d_in_data = d_in->mutable_data<T>(context.GetPlace());

    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    set_zero(context.template device_context<platform::CUDADeviceContext>(),
             d_in, static_cast<T>(0));

    const int pad_left = pads[0];
    const int pad_top = pads[2];
    const int pad_front = pads[4];

    const int num = d_in_dims[0];

    auto stream = context.cuda_device_context().stream();
    int block = PADDLE_CUDA_NUM_THREADS;
    const int out_size = d_out->numel();
    const int in_size = d_in->numel();
    int grid = (out_size + block - 1) / block;

    if (data_format == "NCDHW") {
      const int channels = d_in_dims[1];
      const int in_depth = d_in_dims[2];
      const int in_height = d_in_dims[3];
      const int in_width = d_in_dims[4];
      const int out_depth = d_out_dims[2];
      const int out_height = d_out_dims[3];
      const int out_width = d_out_dims[4];

      if (mode == "reflect") {
        Pad3DGradReflectNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      } else if (mode == "replicate") {
        Pad3DGradReplicateNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      } else if (mode == "circular") {
        Pad3DGradCircularNCDHW<T><<<grid, block, 0, stream>>>(
            out_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      } else {
        grid = (in_size + block - 1) / block;
        Pad3DGradConstNCDHW<T><<<grid, block, 0, stream>>>(
            in_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      }
    } else {
      const int channels = d_in_dims[4];
      const int in_depth = d_in_dims[1];
      const int in_height = d_in_dims[2];
      const int in_width = d_in_dims[3];
      const int out_depth = d_out_dims[1];
      const int out_height = d_out_dims[2];
      const int out_width = d_out_dims[3];
      if (mode == "reflect") {
        Pad3DGradReflectNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      } else if (mode == "replicate") {
        Pad3DGradReplicateNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      } else if (mode == "circular") {
        Pad3DGradCircularNDHWC<T><<<grid, block, 0, stream>>>(
            out_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      } else {
        grid = (in_size + block - 1) / block;
        Pad3DGradConstNDHWC<T><<<grid, block, 0, stream>>>(
            in_size, d_in_data, num, channels, in_depth, in_height, in_width,
            out_depth, out_height, out_width, pad_front, pad_top, pad_left,
            d_out_data);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(pad3d, ops::Pad3dCUDAKernel<plat::float16>,
                        ops::Pad3dCUDAKernel<float>,
                        ops::Pad3dCUDAKernel<double>, ops::Pad3dCUDAKernel<int>,
                        ops::Pad3dCUDAKernel<int64_t>);
REGISTER_OP_CUDA_KERNEL(pad3d_grad, ops::Pad3dGradCUDAKernel<plat::float16>,
                        ops::Pad3dGradCUDAKernel<float>,
                        ops::Pad3dGradCUDAKernel<double>);
