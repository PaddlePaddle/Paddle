#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "paddle/fluid/operators/isfinite_op.h"
#include "paddle/fluid/platform/float16.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    isinf, ops::OverflowKernel<paddle::platform::CUDADeviceContext, int,
                               ops::InfinityFunctor>,
    ops::OverflowKernel<paddle::platform::CUDADeviceContext, float,
                        ops::InfinityFunctor>,
    ops::OverflowKernel<paddle::platform::CUDADeviceContext, double,
                        ops::InfinityFunctor>,
    ops::OverflowKernel<paddle::platform::CUDADeviceContext, plat::float16,
                        ops::InfinityFunctor>);

REGISTER_OP_CUDA_KERNEL(isnan,
                        ops::OverflowKernel<paddle::platform::CUDADeviceContext,
                                            int, ops::NANFunctor>,
                        ops::OverflowKernel<paddle::platform::CUDADeviceContext,
                                            float, ops::NANFunctor>,
                        ops::OverflowKernel<paddle::platform::CUDADeviceContext,
                                            double, ops::NANFunctor>,
                        ops::OverflowKernel<paddle::platform::CUDADeviceContext,
                                            plat::float16, ops::NANFunctor>);

REGISTER_OP_CUDA_KERNEL(
    isfinite, ops::OverflowKernel<paddle::platform::CUDADeviceContext, int,
                                  ops::IsfiniteFunctor>,
    ops::OverflowKernel<paddle::platform::CUDADeviceContext, float,
                        ops::IsfiniteFunctor>,
    ops::OverflowKernel<paddle::platform::CUDADeviceContext, double,
                        ops::IsfiniteFunctor>,
    ops::OverflowKernel<paddle::platform::CUDADeviceContext, plat::float16,
                        ops::IsfiniteFunctor>);
