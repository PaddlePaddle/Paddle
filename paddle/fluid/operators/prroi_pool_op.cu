#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/prroi_pool_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
DEVICE void PrRoIPoolingDistributeDiffCUDA(T* diff, const T top_diff,
                                           const int h, const int w,
                                           const int height, const int width,
                                           const T coeff) {
  bool overflow = (h < 0) || (w < 0) || (h >= height) || (w >= width);
  if (!overflow) {
    paddle::platform::CudaAtomicAdd(diff + h * width + w, top_diff * coeff);
  }
}

template <typename T>
__global__ void GPUPRROIPoolForward(
    const int nthreads, const T* input_data, const T* input_rois,
    const float spatial_scale, const int input_channels, const int height,
    const int width, const int output_channels, const int pooled_height,
    const int pooled_width, const int* rois_batch_id_data, T* output_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    // The output is in order (n, c, ph, pw)
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % output_channels;
    int n = i / pooled_width / pooled_height / output_channels;

    // set roi_batch_id
    int roi_batch_id = rois_batch_id_data[n];

    // [start, end) interval for spatial sampling
    const T* offset_input_rois = input_rois + n * 4;
    T roi_start_w = static_cast<T>(offset_input_rois[0]) * spatial_scale;
    T roi_start_h = static_cast<T>(offset_input_rois[1]) * spatial_scale;
    T roi_end_w = static_cast<T>(offset_input_rois[2]) * spatial_scale;
    T roi_end_h = static_cast<T>(offset_input_rois[3]) * spatial_scale;

    T roi_width = max(roi_end_w - roi_start_w, static_cast<T>(0.0));
    T roi_height = max(roi_end_h - roi_start_h, static_cast<T>(0.0));

    // Compute w and h at input feature map
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    T win_start_w = roi_start_w + bin_size_w * pw;
    T win_start_h = roi_start_h + bin_size_h * ph;
    T win_end_w = win_start_w + bin_size_w;
    T win_end_h = win_start_h + bin_size_h;

    T win_size = max(static_cast<T>(0.0), bin_size_w * bin_size_h);
    int input_channel = (c * pooled_height + ph) * pooled_width + pw;
    const T* offset_input_data =
        input_data +
        (roi_batch_id * input_channels + input_channel) * height * width;

    if (win_size > static_cast<T>(0.0)) {
      int s_w = floor(win_start_w);
      int e_w = ceil(win_end_w);
      int s_h = floor(win_start_h);
      int e_h = ceil(win_end_h);
      T sum_out = 0;

      for (int w_iter = s_w; w_iter < e_w; ++w_iter) {
        for (int h_iter = s_h; h_iter < e_h; ++h_iter) {
          sum_out += PrRoIPoolingMatCalculation(
              offset_input_data, h_iter, w_iter, h_iter + 1, w_iter + 1,
              max(win_start_h, static_cast<T>(h_iter)),
              max(win_start_w, static_cast<T>(w_iter)),
              min(win_end_h, static_cast<T>(h_iter) + static_cast<T>(1.0)),
              min(win_end_w, static_cast<T>(w_iter) + static_cast<T>(1.0)),
              height, width);
        }
      }
      output_data[i] = sum_out / win_size;
    } else {
      output_data[i] = 0.;
    }
  }
}

template <typename T>
__global__ void GPUPRROIPoolBackward(
    const int nthreads, const T* input_rois, const T* output_grad_data,
    const float spatial_scale, const int input_channels, const int height,
    const int width, const int output_channels, const int pooled_height,
    const int pooled_width, const int* rois_batch_id_data, T* input_grad_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    // The output is in order (n, c, ph, pw)
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % output_channels;
    int n = i / pooled_width / pooled_height / output_channels;

    // set roi_batch_id
    int roi_batch_id = rois_batch_id_data[n];
    int input_channel = (c * pooled_height + ph) * pooled_width + pw;
    int input_offset =
        (roi_batch_id * input_channels + input_channel) * height * width;
    T* offset_input_grad_data = input_grad_data + input_offset;
    const T* offset_output_grad_data = output_grad_data + i;

    // [start, end) interval for spatial sampling
    const T* offset_input_rois = input_rois + n * 4;
    T roi_start_w = static_cast<T>(offset_input_rois[0]) * spatial_scale;
    T roi_start_h = static_cast<T>(offset_input_rois[1]) * spatial_scale;
    T roi_end_w = static_cast<T>(offset_input_rois[2]) * spatial_scale;
    T roi_end_h = static_cast<T>(offset_input_rois[3]) * spatial_scale;

    T roi_width = max(roi_end_w - roi_start_w, static_cast<T>(0.0));
    T roi_height = max(roi_end_h - roi_start_h, static_cast<T>(0.0));

    // Compute w and h at input feature map
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    T win_start_w = roi_start_w + bin_size_w * pw;
    T win_start_h = roi_start_h + bin_size_h * ph;
    T win_end_w = win_start_w + bin_size_w;
    T win_end_h = win_start_h + bin_size_h;

    T win_size = max(static_cast<T>(0.0), bin_size_w * bin_size_h);
    int s_w = floor(win_start_w);
    int e_w = ceil(win_end_w);
    int s_h = floor(win_start_h);
    int e_h = ceil(win_end_h);

    T sum_out = win_size == static_cast<T>(0.)
                    ? static_cast<T>(0.)
                    : *offset_output_grad_data / win_size;

    for (int w_iter = s_w; w_iter < e_w; ++w_iter) {
      for (int h_iter = s_h; h_iter < e_h; ++h_iter) {
        PrRoIPoolingMatDistributeDiff(
            offset_input_grad_data, sum_out, h_iter, w_iter, h_iter + 1,
            w_iter + 1, max(win_start_h, static_cast<T>(h_iter)),
            max(win_start_w, static_cast<T>(w_iter)),
            min(win_end_h, static_cast<T>(h_iter) + static_cast<T>(1.0)),
            min(win_end_w, static_cast<T>(w_iter) + static_cast<T>(1.0)),
            height, width, PrRoIPoolingDistributeDiffCUDA<T>);
      }
    }
  }
}

template <typename T>
class GPUPRROIPoolOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");
    auto* out = ctx.Output<Tensor>("Out");

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto output_channels = ctx.Attr<int>("output_channels");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    auto in_dims = in->dims();
    int batch_size = in_dims[0];
    int input_channels = in_dims[1];
    int height = in_dims[2];
    int width = in_dims[3];

    PADDLE_ENFORCE_EQ(input_channels,
                      output_channels * pooled_height * pooled_width,
                      "the channels of input X should equal the product of "
                      "output_channels x pooled_height x pooled_width");

    int rois_num = rois->dims()[0];
    if (rois_num == 0) return;

    auto rois_lod = rois->lod().back();
    int rois_batch_size = rois_lod.size() - 1;
    PADDLE_ENFORCE_EQ(
        rois_batch_size, batch_size,
        "The rois_batch_size and input(X) batch_size must be the same.");
    int rois_num_with_lod = rois_lod[rois_batch_size];
    PADDLE_ENFORCE_EQ(rois_num, rois_num_with_lod,
                      "The rois_num from input and lod must be the same.");

    // set rois batch id
    framework::Tensor rois_batch_id_list;
    rois_batch_id_list.Resize({rois_num});
    int* rois_batch_id_data =
        rois_batch_id_list.mutable_data<int>(platform::CPUPlace());
    for (int n = 0; n < rois_batch_size; ++n) {
      for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
        rois_batch_id_data[i] = n;
      }
    }

    framework::Tensor rois_batch_id_list_gpu;
    framework::TensorCopy(rois_batch_id_list, ctx.GetPlace(),
                          ctx.device_context(), &rois_batch_id_list_gpu);

    int output_size = out->numel();
    int blocks = NumBlocks(output_size);
    int threads = kNumCUDAThreads;

    // call cuda kernel function
    GPUPRROIPoolForward<
        T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        output_size, in->data<T>(), rois->data<T>(), spatial_scale,
        input_channels, height, width, output_channels, pooled_height,
        pooled_width, rois_batch_id_list_gpu.data<int>(),
        out->mutable_data<T>(ctx.GetPlace()));
  }
};

template <typename DeviceContext, typename T>
class GPUPRROIPoolGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");

    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto output_channels = ctx.Attr<int>("output_channels");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    int rois_num = rois->dims()[0];
    int input_channels = in->dims()[1];
    int height = in->dims()[2];
    int width = in->dims()[3];

    if (input_grad) {
      // set roi batch id
      framework::Tensor rois_batch_id_list;
      rois_batch_id_list.Resize({rois_num});
      int* rois_batch_id_data =
          rois_batch_id_list.mutable_data<int>(platform::CPUPlace());
      auto rois_lod = rois->lod().back();
      int rois_batch_size = rois_lod.size() - 1;
      for (int n = 0; n < rois_batch_size; ++n) {
        for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
          rois_batch_id_data[i] = n;
        }
      }

      framework::Tensor rois_batch_id_list_gpu;
      framework::TensorCopy(rois_batch_id_list, ctx.GetPlace(),
                            ctx.device_context(), &rois_batch_id_list_gpu);

      input_grad->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<DeviceContext, T> set_zero;
      set_zero(ctx.cuda_device_context(), input_grad, static_cast<T>(0));

      int output_grad_size = output_grad->numel();
      int blocks = NumBlocks(output_grad_size);
      int threads = kNumCUDAThreads;

      if (output_grad_size > 0) {
        GPUPRROIPoolBackward<
            T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
            output_grad_size, rois->data<T>(), output_grad->data<T>(),
            spatial_scale, input_channels, height, width, output_channels,
            pooled_height, pooled_width, rois_batch_id_list_gpu.data<int>(),
            input_grad->mutable_data<T>(ctx.GetPlace()));
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(prroi_pool, ops::GPUPRROIPoolOpKernel<float>,
                        ops::GPUPRROIPoolOpKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    prroi_pool_grad,
    ops::GPUPRROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUPRROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, double>);
