#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/fluid/operators/adagrad_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

namespace {

template <typename T, int block_size>
__global__ void SparseAdagradFunctorKernel(const T* grad, const int64_t* rows,
                                           const T* learning_rate, T* param,
                                           T* moment, int64_t row_numel,
                                           T epsilon) {
  const int ty = blockIdx.x;
  int tid = threadIdx.x;

  grad += ty * row_numel;
  param += rows[ty] * row_numel;
  moment += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(param + index,
                                    -1.0 * learning_rate[0] * grad[index] /
                                        (sqrt(moment[index]) + epsilon));
  }
}
}  // namespace

template <typename T>
struct SparseAdagradFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& grad,
                  const framework::Tensor& learning_rate, T epsilon,
                  framework::Tensor* moment, framework::Tensor* param) {
    // 1. g_m.rows = set(g.rows)
    auto grad_width = grad.value().dims()[1];
    math::scatter::MergeAdd<platform::CUDADeviceContext, T> merge_func;
    auto grad_merge = merge_func(context, grad);
    auto* grad_merge_data = grad_merge.mutable_value()->template data<T>();
    framework::Vector<int64_t> merge_rows(grad_merge.rows());
    // 2. m += g_m * g_m
    math::scatter::Mul<platform::CUDADeviceContext, T> sqare_func;
    auto grad_square = sqare_func(context, grad_merge, grad_merge);

    math::SelectedRowsAddToTensor<platform::CUDADeviceContext, T> functor;
    functor(context, grad_square, moment);

    // 3. update parameter
    auto* lr = learning_rate.data<T>();
    auto* param_data = param->data<T>();
    auto* moment_data = moment->data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid2(merge_rows.size(), 1);
    SparseAdagradFunctorKernel<
        T, block_size><<<grid2, threads, 0, context.stream()>>>(
        grad_merge_data, merge_rows.CUDAMutableData(context.GetPlace()), lr,
        param_data, moment_data, grad_width, epsilon);
  }
};

template struct SparseAdagradFunctor<platform::CUDADeviceContext, float>;
template struct SparseAdagradFunctor<platform::CUDADeviceContext, double>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    adagrad, ops::AdagradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::AdagradOpKernel<paddle::platform::CUDADeviceContext, double>);
