#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/allclose_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct GetTensorValue<platform::CUDADeviceContext, T> {
  T operator()(const platform::CUDADeviceContext& dev_ctx,
               const framework::Tensor& tensor) const {
    const T* data = tensor.data<T>();
    T value;
    const auto gpu_place =
        BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace());
    memory::Copy(platform::CPUPlace(), &value, gpu_place, data, sizeof(T),
                 dev_ctx.stream());
    return value;
  }
};

template <typename T>
__global__ void AllcloseCUDAKernel(const T* in_data, const T* other_data,
                                   const double rtol, const double atol,
                                   bool equal_nan, int num, bool* out_data) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  bool val;
  for (int i = idx; i < num; i += blockDim.x * gridDim.x) {
    const T a = in_data[i], b = other_data[i];
    if (isnan(a) || isnan(b)) {
      val = equal_nan && isnan(a) == isnan(b);
    } else {
      T left = (a > b ? a - b : b - a);
      T right = atol + (b > 0 ? rtol * b : (-rtol) * b);
      T diff = (left > right ? left - right : right - left);
      val = a == b || left <= right || diff <= 1e-15;
    }
    if (!val) *out_data = false;
  }
}

template <typename T>
struct AllcloseFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor& in, const framework::Tensor& other,
                  const double rtol, const double atol, bool equal_nan,
                  framework::Tensor* output) {
    int num = in.numel();
    const T* in_data = in.data<T>();
    const T* other_data = other.data<T>();
    bool* out_data = output->mutable_data<bool>(dev_ctx.GetPlace());
    int block = 1024;
    int grid = (block - 1 + num) / block;
    grid = (grid > block) ? block : grid;
#ifdef PADDLE_WITH_HIP
    hipMemset(out_data, true, sizeof(bool));
#else
    hipMemset(out_data, true, sizeof(bool));
#endif
    AllcloseCUDAKernel<T><<<grid, block, 0, dev_ctx.stream()>>>(
        in_data, other_data, rtol, atol, equal_nan, num, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(allclose, ops::AllcloseKernel<CUDA, float>,
                        ops::AllcloseKernel<CUDA, double>);
