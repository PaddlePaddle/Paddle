#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/range_op.h"
#include "paddle/fluid/operators/utils.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void RangeKernel(T start, T step, int64_t size, T* out) {
  CUDA_KERNEL_LOOP(index, size) { out[index] = start + step * index; }
}

template <typename T>
class CUDARangeKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* start_t = context.Input<framework::Tensor>("Start");
    auto* end_t = context.Input<framework::Tensor>("End");
    auto* step_t = context.Input<framework::Tensor>("Step");
    auto* out = context.Output<framework::Tensor>("Out");

    T start = GetValue<T>(start_t);
    T end = GetValue<T>(end_t);
    T step = GetValue<T>(step_t);

    int64_t size = 0;
    GetSize(start, end, step, &size);
    out->Resize(framework::make_ddim({size}));
    T* out_data = out->mutable_data<T>(context.GetPlace());

    auto stream = context.cuda_device_context().stream();
    int block = std::min(size, static_cast<int64_t>(256));
    int grid = (size + block - 1) / block;
    RangeKernel<T><<<grid, block, 0, stream>>>(start, step, size, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(range, ops::CUDARangeKernel<int>,
                        ops::CUDARangeKernel<int64_t>,
                        ops::CUDARangeKernel<float>,
                        ops::CUDARangeKernel<double>);
