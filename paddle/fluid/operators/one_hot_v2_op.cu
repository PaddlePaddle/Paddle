#include "hip/hip_runtime.h"
//   Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/one_hot_v2_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;

template <typename InT, typename OutT>
__global__ void FillOutputKernel(const InT* p_in_data, OutT* p_out_data,
                                 const int64_t numel, const int depth) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel && p_in_data[idx] >= 0 && p_in_data[idx] < depth) {
    *(p_out_data + (idx * depth) + p_in_data[idx]) = 1.0;
  }
}

template <typename DeviceContext, typename InT>
struct OneHotV2OpCUDAFunctor {
  const framework::LoDTensor* in_;
  framework::LoDTensor* out_;
  const DeviceContext& ctx_;
  int depth_;

  OneHotV2OpCUDAFunctor(const framework::LoDTensor* in,
                        framework::LoDTensor* out, int depth,
                        const DeviceContext& ctx)
      : in_(in), out_(out), depth_(depth), ctx_(ctx) {}

  template <typename OutT>
  void apply() const {
    auto* p_in_data = in_->data<InT>();
    auto numel = in_->numel();
    auto* p_out_data = out_->mutable_data<OutT>(ctx_.GetPlace());
    auto stream = ctx_.stream();
    math::set_constant(ctx_, out_, 0.0);

    FillOutputKernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                           PADDLE_CUDA_NUM_THREADS,
                       PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        p_in_data, p_out_data, numel, depth_);
  }
};

using LoDTensor = framework::LoDTensor;
template <typename DeviceContext, typename T>
class OneHotV2CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");

    int depth = -1;
    if (context.HasInput("depth_tensor")) {
      auto* depth_tensor = context.Input<framework::Tensor>("depth_tensor");
      if (platform::is_gpu_place(depth_tensor->place())) {
        framework::Tensor temp;
        TensorCopySync(*depth_tensor, platform::CPUPlace(), &temp);
        depth = *temp.data<int32_t>();
      } else {
        depth = *depth_tensor->data<int32_t>();
      }

      auto out_dims = out->dims();
      out_dims[out_dims.size() - 1] = depth;
      out->Resize(out_dims);
    } else {
      depth = context.Attr<int>("depth");
    }
    framework::VisitDataType(
        static_cast<framework::proto::VarType::Type>(
            context.Attr<int>("dtype")),
        OneHotV2OpCUDAFunctor<DeviceContext, T>(
            in, out, depth, context.template device_context<DeviceContext>()));
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    one_hot_v2,
    ops::OneHotV2CUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::OneHotV2CUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
