#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/mode_op.h"
#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/fluid/operators/top_k_v2_op.h"

namespace paddle {
namespace operators {

int ComputeBlockSize(int col) {
  if (col > 512)
    return 1024;
  else if (col > 256 && col <= 512)
    return 512;
  else if (col > 128 && col <= 256)
    return 256;
  else if (col > 64 && col <= 128)
    return 128;
  else
    return 64;
}

template <typename T>
void getModebySort(const platform::CUDADeviceContext& ctx,
                   const framework::Tensor* input_tensor,
                   const int64_t num_cols, const int64_t num_rows,
                   T* out_tensor, int64_t* indices_tensor) {
  framework::Tensor input_tmp;
  framework::TensorCopy(*input_tensor, ctx.GetPlace(), &input_tmp);
  T* input_tmp_data = input_tmp.mutable_data<T>(ctx.GetPlace());
  input_tmp.Resize(phi::make_ddim({num_rows, num_cols}));
  thrust::device_ptr<T> out_tensor_ptr(out_tensor);
  thrust::device_ptr<int64_t> indices_tensor_ptr(indices_tensor);

  for (int64_t i = 0; i < num_rows; ++i) {
    T* begin = input_tmp_data + num_cols * i;
    T* end = input_tmp_data + num_cols * (i + 1);
    thrust::device_vector<int64_t> indices_data(num_cols);
    thrust::sequence(thrust::device, indices_data.begin(),
                     indices_data.begin() + num_cols);
    thrust::sort_by_key(thrust::device, begin, end, indices_data.begin());
    int unique = 1 + thrust::inner_product(thrust::device, begin, end - 1,
                                           begin + 1, 0, thrust::plus<int>(),
                                           thrust::not_equal_to<T>());
    thrust::device_vector<T> keys_data(unique);
    thrust::device_vector<int64_t> cnts_data(unique);
    thrust::reduce_by_key(thrust::device, begin, end,
                          thrust::constant_iterator<int>(1), keys_data.begin(),
                          cnts_data.begin());
    auto it = thrust::max_element(thrust::device, cnts_data.begin(),
                                  cnts_data.begin() + unique);
    T mode = keys_data[it - cnts_data.begin()];
    int64_t counts = cnts_data[it - cnts_data.begin()];
    auto pos = thrust::find(thrust::device, begin, end, mode);
    int64_t index = indices_data[pos - begin + counts - 1];
    out_tensor_ptr[i] = static_cast<T>(mode);
    indices_tensor_ptr[i] = static_cast<int64_t>(index);
  }
}

template <typename DeviceContext, typename T>
class ModeOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* input = ctx.Input<framework::Tensor>("X");
    auto* output = ctx.Output<framework::Tensor>("Out");
    auto* indices = ctx.Output<framework::Tensor>("Indices");
    int axis = static_cast<int>(ctx.Attr<int>("axis"));
    bool keepdim = static_cast<bool>(ctx.Attr<bool>("keepdim"));

    // get the input dims
    const auto& in_dims = input->dims();
    // calcluate the real axis
    if (axis < 0) axis += in_dims.size();

    auto out_dims = output->dims();

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    int64_t* indices_data = indices->mutable_data<int64_t>(ctx.GetPlace());

    if (axis == in_dims.size() - 1) {
      const int64_t& input_height =
          phi::product(phi::slice_ddim(in_dims, 0, in_dims.size() - 1));
      const int64_t& input_width = in_dims[in_dims.size() - 1];
      const auto& dev_ctx = ctx.cuda_device_context();
      getModebySort<T>(dev_ctx, input, input_width, input_height, output_data,
                       indices_data);
    } else {
      std::vector<int> trans_axis;
      for (int i = 0; i < axis; i++) {
        trans_axis.emplace_back(i);
      }
      trans_axis.emplace_back(in_dims.size() - 1);
      for (int i = axis + 1; i < in_dims.size() - 1; i++) {
        trans_axis.emplace_back(i);
      }
      trans_axis.emplace_back(axis);

      if (!keepdim) {
        std::vector<int> tmp_out_shape;
        for (int i = 0; i < axis; i++) {
          tmp_out_shape.emplace_back(in_dims[i]);
        }
        tmp_out_shape.emplace_back(1);
        for (int i = axis + 1; i < in_dims.size(); i++) {
          tmp_out_shape.emplace_back(in_dims[i]);
        }
        framework::DDim tmp_out_dim = phi::make_ddim(tmp_out_shape);
        output->Resize(tmp_out_dim);
        indices->Resize(tmp_out_dim);
      }

      framework::DDim trans_shape(in_dims);
      framework::DDim trans_out_shape(in_dims);
      for (int i = 0; i < trans_axis.size(); i++) {
        trans_shape[i] = in_dims[trans_axis[i]];
        trans_out_shape[i] = in_dims[trans_axis[i]];
      }
      trans_out_shape[in_dims.size() - 1] = 1;

      // second step, tranpose the input
      framework::Tensor trans_input;
      trans_input.mutable_data<T>(trans_shape, ctx.GetPlace());
      int ndims = trans_axis.size();
      const auto& dev_ctx = ctx.cuda_device_context();
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, *input,
                                                   &trans_input, trans_axis);
      framework::Tensor trans_ind;
      int64_t* trans_ind_data =
          trans_ind.mutable_data<int64_t>(trans_out_shape, ctx.GetPlace());
      framework::Tensor trans_out;
      T* trans_out_data =
          trans_out.mutable_data<T>(trans_out_shape, ctx.GetPlace());

      const int64_t input_height =
          phi::product(phi::slice_ddim(trans_shape, 0, trans_shape.size() - 1));
      const int64_t input_width = trans_shape[trans_shape.size() - 1];
      getModebySort<T>(dev_ctx, &trans_input, input_width, input_height,
                       trans_out_data, trans_ind_data);
      // last step, tranpose back the indices and output
      TransCompute<platform::CUDADeviceContext, int64_t>(
          ndims, dev_ctx, trans_ind, indices, trans_axis);
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, trans_out,
                                                   output, trans_axis);
      if (!keepdim) {
        output->Resize(out_dims);
        indices->Resize(out_dims);
      }
    }
  }
};

template <typename DeviceContext, typename T>
class ModeOpGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* x = context.Input<framework::Tensor>("X");
    auto* out_grad =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* indices = context.Input<framework::Tensor>("Indices");
    auto* x_grad =
        context.Output<framework::Tensor>(framework::GradVarName("X"));
    int axis = context.Attr<int>("axis");

    const auto& in_dims = x->dims();
    auto out_dims = indices->dims();

    if (axis < 0) axis += in_dims.size();
    // allocate the cuda memory for the x_grad
    T* x_grad_data = x_grad->mutable_data<T>(context.GetPlace());
    const T* out_grad_data = out_grad->data<T>();
    const int64_t* indices_data = indices->data<int64_t>();

    int pre, n, post;
    GetDims(in_dims, axis, &pre, &n, &post);

    // calcluate the block and grid num
    auto& dev_ctx = context.cuda_device_context();
    int block_size = ComputeBlockSize(post);
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
    int grid_size = std::min(max_blocks, pre);
    AssignGradWithAxis<T><<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
        out_grad_data, indices_data, x_grad_data, pre, post, n, 1);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    mode, ops::ModeOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ModeOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ModeOpCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ModeOpCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    mode_grad,
    ops::ModeOpGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::ModeOpGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::ModeOpGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::ModeOpGradCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
