#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/fake_quantize_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void FindAbsMaxKernel(const T* in, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ T shared_max_data[];
  if (gridDim.x > 1) {
    shared_max_data[tid] = T(0);
    for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
      T tmp = fabs(in[i]);
      if (tmp > shared_max_data[tid]) {
        shared_max_data[tid] = tmp;
      }
    }
  } else {
    if (bid < n) {
      shared_max_data[tid] = fabs(in[bid]);
    } else {
      shared_max_data[tid] = T(0);
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
struct FindAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx, const T* in,
                  const int num, T* out) {
    int block = 1024;
    int grid = (block - 1 + num) / block;
    grid = (grid > block) ? block : grid;

    framework::Tensor max;
    T* max_data =
        max.mutable_data<T>(framework::make_ddim({grid}), ctx.GetPlace());
    FindAbsMaxKernel<T><<<grid, block, 1024 * sizeof(T), ctx.stream()>>>(
        in, num, max_data);
    FindAbsMaxKernel<T><<<1, block, 1024 * sizeof(T), ctx.stream()>>>(
        max_data, grid, out);
  }
};

template struct FindAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
__global__ void FindChannelAbsMaxKernel(const T* in, const int n, const int c,
                                        T* out) {
  int tid = threadIdx.x;
  int channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  extern __shared__ T shared_max_data[];
  shared_max_data[tid] = T(0);
  for (int i = tid; i < channel_size; i += blockDim.x) {
    T tmp = fabs(in_c[i]);
    if (tmp > shared_max_data[tid]) {
      shared_max_data[tid] = tmp;
    }
  }
  __syncthreads();
  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
struct FindChannelAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx, const T* in,
                  const int num, const int channel, T* out) {
    int block = 1024;
    int grid = channel;
    FindChannelAbsMaxKernel<T><<<grid, block, 1024 * sizeof(T), ctx.stream()>>>(
        in, num, channel, out);
  }
};

template struct FindChannelAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
__global__ void ClipAndQuantKernel(const T* in, const T* scale,
                                   const int bin_cnt, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  T s = scale[0];
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    T x = in[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt / s * v;
    out[i] = round(v);
  }
}

template <typename T>
__global__ void ClipAndQuantDequantKernel(const T* in, const T* scale,
                                          const int bin_cnt, const int n,
                                          T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  T s = scale[0];
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    T x = in[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt / s * v;
    out[i] = round(v) * s / bin_cnt;
  }
}

template <typename T>
struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = (block - 1 + num) / block;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ClipAndQuantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, out_data);
  }
};

template struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, float>;

template <typename T>
struct ClipAndFakeQuantDequantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = (block - 1 + num) / block;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ClipAndQuantDequantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, out_data);
  }
};

template struct ClipAndFakeQuantDequantFunctor<platform::CUDADeviceContext,
                                               float>;

template <typename T>
__global__ void ChannelClipAndQuantKernel(const T* in, const T* scale,
                                          const int bin_cnt, const int n,
                                          const int c, T* out) {
  int tid = threadIdx.x;

  int channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  T* out_c = out + blockIdx.x * channel_size;

  T s = scale[blockIdx.x];
  for (int i = tid; i < channel_size; i += blockDim.x) {
    T x = in_c[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt / s * v;
    out_c[i] = round(v);
  }
}

template <typename T>
struct ChannelClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, const int channel,
                  framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = channel;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ChannelClipAndQuantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, channel, out_data);
  }
};

template struct ChannelClipAndFakeQuantFunctor<platform::CUDADeviceContext,
                                               float>;

template <typename T>
__global__ void FindRangeAbsMaxAndFillArray(const T* cur_scale,
                                            const T* last_scale,
                                            const int64_t* iter,
                                            const int window_size, T* scale_arr,
                                            T* out_scale, int* need_find_max,
                                            int* out_size) {
  int it = iter[0];
  int idx = it % window_size;
  T removed = scale_arr[idx];
  T cur = cur_scale[0];
  scale_arr[idx] = cur;
  T max = last_scale[0];
  out_scale[0] = max < cur ? cur : max;
  if (fabs(removed - max) < 1e-6) {
    need_find_max[0] = 1;
    out_size[0] = it > window_size ? window_size : it;
  } else {
    need_find_max[0] = 0;
  }
}

template <typename T>
struct FindRangeAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& cur_scale,
                  const framework::Tensor& last_scale,
                  const framework::Tensor& iter, const int window_size,
                  framework::Tensor* scales_arr, framework::Tensor* out_scale) {
    const auto gpu_place = boost::get<platform::CUDAPlace>(ctx.GetPlace());

    T* scale_arr = scales_arr->mutable_data<T>(gpu_place);
    T* out_scale_data = out_scale->mutable_data<T>(gpu_place);

    framework::Tensor need_find_max, out_size;
    int* find_max = need_find_max.mutable_data<int>({1}, gpu_place);
    int* out_size_data = out_size.mutable_data<int>({1}, gpu_place);

    FindRangeAbsMaxAndFillArray<T><<<1, 1, 0, ctx.stream()>>>(
        cur_scale.data<T>(), last_scale.data<T>(), iter.data<int64_t>(),
        window_size, scale_arr, out_scale_data, find_max, out_size_data);

    int g_find_max;
    memory::Copy(platform::CPUPlace(), &g_find_max, gpu_place, find_max,
                 sizeof(int), ctx.stream());
    ctx.Wait();
    if (g_find_max) {
      int len;
      memory::Copy(platform::CPUPlace(), &len, gpu_place, out_size_data,
                   sizeof(int), ctx.stream());
      ctx.Wait();
      FindAbsMaxFunctor<platform::CUDADeviceContext, T>()(ctx, scale_arr, len,
                                                          out_scale_data);
    }
  }
};

template struct FindRangeAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
struct FindMovingAverageAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in_accum,
                  const framework::Tensor& in_state, const T* cur_scale,
                  const float rate, framework::Tensor* out_state,
                  framework::Tensor* out_accum, framework::Tensor* out_scale) {
    const auto gpu_place = boost::get<platform::CUDAPlace>(ctx.GetPlace());

    T accum;
    T state;
    T scale;
    memory::Copy(platform::CPUPlace(), &accum, gpu_place, in_accum.data<T>(),
                 sizeof(T), ctx.stream());
    memory::Copy(platform::CPUPlace(), &state, gpu_place, in_state.data<T>(),
                 sizeof(T), ctx.stream());
    memory::Copy(platform::CPUPlace(), &scale, gpu_place, cur_scale, sizeof(T),
                 ctx.stream());
    ctx.Wait();
    state = rate * state + 1;
    accum = rate * accum + scale;
    scale = accum / state;

    memory::Copy(gpu_place, out_accum->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &accum, sizeof(T), ctx.stream());
    memory::Copy(gpu_place, out_state->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &state, sizeof(T), ctx.stream());
    memory::Copy(gpu_place, out_scale->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &scale, sizeof(T), ctx.stream());
    ctx.Wait();
  }
};

template struct FindMovingAverageAbsMaxFunctor<platform::CUDADeviceContext,
                                               float>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(fake_quantize_abs_max,
                        ops::FakeQuantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_channel_wise_quantize_abs_max,
                        ops::FakeChannelWiseQuantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_range_abs_max,
                        ops::FakeQuantizeRangeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_moving_average_abs_max,
    ops::FakeQuantizeMovingAverageAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(moving_average_abs_max_scale,
                        ops::MovingAverageAbsMaxScaleKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_dequantize_moving_average_abs_max,
    ops::FakeQuantizeDequantizeMovingAverageAbsMaxKernel<CUDA, float>);
