#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/fake_quantize_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void FindAbsMaxKernel(const T* in, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ T shared_max_data[];
  if (gridDim.x > 1) {
    shared_max_data[tid] = T(0);
    for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
      T tmp = fabs(in[i]);
      if (tmp > shared_max_data[tid]) {
        shared_max_data[tid] = tmp;
      }
    }
  } else {
    if (bid < n) {
      shared_max_data[tid] = fabs(in[bid]);
    } else {
      shared_max_data[tid] = T(0);
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
struct FindAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx, const T* in,
                  const int num, T* out) {
    int block = 1024;
    int grid = (block - 1 + num) / block;
    grid = (grid > block) ? block : grid;

    framework::Tensor max;
    T* max_data =
        max.mutable_data<T>(framework::make_ddim({grid}), ctx.GetPlace());
    FindAbsMaxKernel<T><<<grid, block, 1024 * sizeof(T), ctx.stream()>>>(
        in, num, max_data);
    FindAbsMaxKernel<T><<<1, block, 1024 * sizeof(T), ctx.stream()>>>(
        max_data, grid, out);
  }
};

template struct FindAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
__global__ void FindChannelAbsMaxKernelQuantAxis0(const T* in, const int n,
                                                  const int c, T* out) {
  int tid = threadIdx.x;
  int channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  extern __shared__ T shared_max_data[];
  shared_max_data[tid] = T(0);
  for (int i = tid; i < channel_size; i += blockDim.x) {
    T tmp = fabs(in_c[i]);
    if (tmp > shared_max_data[tid]) {
      shared_max_data[tid] = tmp;
    }
  }
  __syncthreads();
  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
__global__ void FindChannelAbsMaxKernelQuantAxis1(const T* in, const int n,
                                                  const int cin, const int cout,
                                                  T* out) {
  extern __shared__ T shared_max_data[];
  int cout_wh_size = n / cin;
  int wh_size = n / (cin * cout);

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  const T* in_current = in + tid * cout_wh_size + bid * wh_size;
  shared_max_data[tid] = T(0);
  for (int i = 0; i < wh_size; i++) {
    T tmp = fabs(in_current[i]);
    if (tmp > shared_max_data[tid]) {
      shared_max_data[tid] = tmp;
    }
  }
  __syncthreads();

  int len = blockDim.x;
  for (int i = (len + 1) / 2; i > 0; len = i, i = (i + 1) / 2) {
    if (tid < i && tid + i < len &&
        shared_max_data[tid] < shared_max_data[tid + i]) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    if (i == 1) {
      i = 0;  // break the loop
    }
    __syncthreads();
  }
  if (tid == 0 && shared_max_data[0] > out[bid]) {
    out[bid] = shared_max_data[0];
  }
}

template <typename T>
struct FindChannelAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in_tensor, const int quant_axis,
                  T* out_abs_max) {
    PADDLE_ENFORCE_EQ(
        quant_axis == 0 || quant_axis == 1, true,
        platform::errors::InvalidArgument("'quant_axis' should be 0 or 1, but "
                                          "the received is %d",
                                          quant_axis));
    const int num = in_tensor.numel();
    auto in_dims = in_tensor.dims();
    const T* in_data = in_tensor.data<T>();
    if (quant_axis == 0) {
      int cout = in_dims[0];
      int grid = cout;
      int block = 1024;
      FindChannelAbsMaxKernelQuantAxis0<
          T><<<grid, block, block * sizeof(T), ctx.stream()>>>(
          in_data, num, cout, out_abs_max);
    } else if (quant_axis == 1) {
      int cin = in_dims[0];
      int cout = in_dims[1];
      int grid = cout;
      int max_threads = 1024;

#ifdef PADDLE_WITH_HIP
      hipMemset(out_abs_max, 0, sizeof(T) * cout);
#else
      hipMemset(out_abs_max, 0, sizeof(T) * cout);
#endif

      for (int i = 0; i < cin / max_threads; i++) {
        int block = max_threads;
        FindChannelAbsMaxKernelQuantAxis1<
            T><<<grid, block, block * sizeof(T), ctx.stream()>>>(
            in_data, num, cin, cout, out_abs_max);
        in_data += num / cin;
      }

      int block = cin % max_threads;
      if (block > 0) {
        FindChannelAbsMaxKernelQuantAxis1<
            T><<<grid, block, block * sizeof(T), ctx.stream()>>>(
            in_data, num, in_dims[0], in_dims[1], out_abs_max);
      }
    }
  }
};

template struct FindChannelAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
__global__ void ClipAndQuantKernel(const T* in, const T* scale,
                                   const int bin_cnt, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  T s = scale[0];
  T inv_s = inverse(s);
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    T x = in[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out[i] = round(v);
  }
}

template <typename T>
__global__ void ClipAndQuantDequantKernel(const T* in, const T* scale,
                                          const int bin_cnt, const int n,
                                          T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  T s = scale[0];
  T inv_s = inverse(s);
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    T x = in[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out[i] = round(v) * s / bin_cnt;
  }
}

template <typename T>
struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = (block - 1 + num) / block;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ClipAndQuantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, out_data);
  }
};

template struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, float>;

template <typename T>
struct ClipAndFakeQuantDequantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = (block - 1 + num) / block;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ClipAndQuantDequantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, out_data);
  }
};

template struct ClipAndFakeQuantDequantFunctor<platform::CUDADeviceContext,
                                               float>;

// ChannelClipAndQuantKernel for quant_axis is 0
template <typename T>
__global__ void ChannelClipAndQuantKernelQuantAxis0(const T* in, const T* scale,
                                                    const int bin_cnt,
                                                    const int n, const int c,
                                                    T* out) {
  int tid = threadIdx.x;

  int channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  T* out_c = out + blockIdx.x * channel_size;

  T s = scale[blockIdx.x];
  T inv_s = inverse(s);

  for (int i = tid; i < channel_size; i += blockDim.x) {
    T x = in_c[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out_c[i] = round(v);
  }
}

// ChannelClipAndQuantKernel for quant_axis is 1
template <typename T>
__global__ void ChannelClipAndQuantKernelQuantAxis1(const T* in, const T* scale,
                                                    const int bin_cnt,
                                                    const int n, const int cin,
                                                    const int cout, T* out) {
  T s = scale[blockIdx.x % cout];
  T inv_s = inverse(s);

  int wh_size = n / (cin * cout);
  const T* in_c = in + blockIdx.x * wh_size;
  T* out_c = out + blockIdx.x * wh_size;

  for (int i = threadIdx.x; i < wh_size; i += blockDim.x) {
    T x = in_c[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out_c[i] = round(v);
  }
}

template <typename T>
struct ChannelClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, const int quant_axis,
                  framework::Tensor* out) {
    PADDLE_ENFORCE_EQ(
        quant_axis == 0 || quant_axis == 1, true,
        platform::errors::InvalidArgument("'quant_axis' should be 0 or 1, but "
                                          "the received is %d",
                                          quant_axis));

    int num = in.numel();
    auto in_dims = in.dims();
    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    if (quant_axis == 0) {
      int grid = in_dims[0];
      int block = 1024;
      ChannelClipAndQuantKernelQuantAxis0<T><<<grid, block, 0, ctx.stream()>>>(
          in_data, scale_data, bin_cnt, num, in_dims[0], out_data);
    } else if (quant_axis == 1) {
      int grid = in_dims[0] * in_dims[1];
      int block = 1024;
      ChannelClipAndQuantKernelQuantAxis1<T><<<grid, block, 0, ctx.stream()>>>(
          in_data, scale_data, bin_cnt, num, in_dims[0], in_dims[1], out_data);
    }
  }
};

template struct ChannelClipAndFakeQuantFunctor<platform::CUDADeviceContext,
                                               float>;

template <typename T>
__global__ void FindRangeAbsMaxAndFillArray(const T* cur_scale,
                                            const T* last_scale,
                                            const int64_t* iter,
                                            const int window_size, T* scale_arr,
                                            T* out_scale, int* need_find_max,
                                            int* out_size) {
  int it = iter[0];
  int idx = it % window_size;
  T removed = scale_arr[idx];
  T cur = cur_scale[0];
  scale_arr[idx] = cur;
  T max = last_scale[0];
  out_scale[0] = max < cur ? cur : max;
  if (fabs(removed - max) < 1e-6) {
    need_find_max[0] = 1;
    out_size[0] = it > window_size ? window_size : it;
  } else {
    need_find_max[0] = 0;
  }
}

template <typename T>
struct FindRangeAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& cur_scale,
                  const framework::Tensor& last_scale,
                  const framework::Tensor& iter, const int window_size,
                  framework::Tensor* scales_arr, framework::Tensor* out_scale) {
    const auto gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());

    T* scale_arr = scales_arr->mutable_data<T>(gpu_place);
    T* out_scale_data = out_scale->mutable_data<T>(gpu_place);

    framework::Tensor need_find_max, out_size;
    int* find_max = need_find_max.mutable_data<int>({1}, gpu_place);
    int* out_size_data = out_size.mutable_data<int>({1}, gpu_place);

    FindRangeAbsMaxAndFillArray<T><<<1, 1, 0, ctx.stream()>>>(
        cur_scale.data<T>(), last_scale.data<T>(), iter.data<int64_t>(),
        window_size, scale_arr, out_scale_data, find_max, out_size_data);

    int g_find_max;
    memory::Copy(platform::CPUPlace(), &g_find_max, gpu_place, find_max,
                 sizeof(int), ctx.stream());
    ctx.Wait();
    if (g_find_max) {
      int len;
      memory::Copy(platform::CPUPlace(), &len, gpu_place, out_size_data,
                   sizeof(int), ctx.stream());
      ctx.Wait();
      FindAbsMaxFunctor<platform::CUDADeviceContext, T>()(ctx, scale_arr, len,
                                                          out_scale_data);
    }
  }
};

template struct FindRangeAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
struct FindMovingAverageAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in_accum,
                  const framework::Tensor& in_state, const T* cur_scale,
                  const float rate, framework::Tensor* out_state,
                  framework::Tensor* out_accum, framework::Tensor* out_scale) {
    const auto gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());

    T accum;
    T state;
    T scale;
    memory::Copy(platform::CPUPlace(), &accum, gpu_place, in_accum.data<T>(),
                 sizeof(T), ctx.stream());
    memory::Copy(platform::CPUPlace(), &state, gpu_place, in_state.data<T>(),
                 sizeof(T), ctx.stream());
    memory::Copy(platform::CPUPlace(), &scale, gpu_place, cur_scale, sizeof(T),
                 ctx.stream());
    ctx.Wait();
    state = rate * state + 1;
    accum = rate * accum + scale;
    scale = accum / state;

    memory::Copy(gpu_place, out_accum->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &accum, sizeof(T), ctx.stream());
    memory::Copy(gpu_place, out_state->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &state, sizeof(T), ctx.stream());
    memory::Copy(gpu_place, out_scale->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &scale, sizeof(T), ctx.stream());
    ctx.Wait();
  }
};

// ChannelClipAndQuantDequantKernel for quant_axis is 0
template <typename T>
__global__ void ChannelClipAndQuantDequantKernelQuantAxis0(
    const T* in, const T* scale, const int bin_cnt, const int n, const int c,
    T* out) {
  int tid = threadIdx.x;

  int channel_size = n / c;
  const T* in_c = in + blockIdx.x * channel_size;
  T* out_c = out + blockIdx.x * channel_size;

  T s = scale[blockIdx.x];
  T inv_s = inverse(s);

  for (int i = tid; i < channel_size; i += blockDim.x) {
    T x = in_c[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out_c[i] = round(v) * s / bin_cnt;
  }
}

// ChannelClipAndQuantDequantKernel for quant_axis is 1
template <typename T>
__global__ void ChannelClipAndQuantDequantKernelQuantAxis1(
    const T* in, const T* scale, const int bin_cnt, const int n, const int cin,
    const int cout, T* out) {
  T s = scale[blockIdx.x % cout];
  T inv_s = inverse(s);

  int wh_size = n / (cin * cout);
  const T* in_c = in + blockIdx.x * wh_size;
  T* out_c = out + blockIdx.x * wh_size;

  for (int i = threadIdx.x; i < wh_size; i += blockDim.x) {
    T x = in_c[i];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt * inv_s * v;
    out_c[i] = round(v) * s / bin_cnt;
  }
}

template <typename T>
struct ChannelClipFakeQuantDequantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, const int quant_axis,
                  framework::Tensor* out) {
    // At present, channelwise quantization supports conv2d, depthwise_conv2d
    // conv2d_transpose and mul
    PADDLE_ENFORCE_EQ(
        quant_axis == 0 || quant_axis == 1, true,
        platform::errors::InvalidArgument("'quant_axis' should be 0 or 1, but "
                                          "the received is %d",
                                          quant_axis));

    int num = in.numel();
    auto in_dims = in.dims();

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    if (quant_axis == 0) {
      int grid = in_dims[0];
      int block = 1024;
      ChannelClipAndQuantDequantKernelQuantAxis0<
          T><<<grid, block, 0, ctx.stream()>>>(in_data, scale_data, bin_cnt,
                                               num, in_dims[0], out_data);
    } else if (quant_axis == 1) {
      int grid = in_dims[0] * in_dims[1];
      int block = 1024;

      ChannelClipAndQuantDequantKernelQuantAxis1<
          T><<<grid, block, 0, ctx.stream()>>>(
          in_data, scale_data, bin_cnt, num, in_dims[0], in_dims[1], out_data);
    }
  }
};

template struct ChannelClipFakeQuantDequantFunctor<platform::CUDADeviceContext,
                                                   float>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(fake_quantize_abs_max,
                        ops::FakeQuantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_dequantize_abs_max,
                        ops::FakeQuantizeDequantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_channel_wise_quantize_abs_max,
                        ops::FakeChannelWiseQuantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_range_abs_max,
                        ops::FakeQuantizeRangeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_moving_average_abs_max,
    ops::FakeQuantizeMovingAverageAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(moving_average_abs_max_scale,
                        ops::MovingAverageAbsMaxScaleKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_dequantize_moving_average_abs_max,
    ops::FakeQuantizeDequantizeMovingAverageAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_dequantize_grad,
                        ops::FakeQuantDequantGradKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(
    fake_channel_wise_quantize_dequantize_abs_max,
    ops::FakeChannelWiseQuantizeDequantizeAbsMaxKernel<CUDA, float>);
