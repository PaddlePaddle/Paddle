#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/operators/fake_quantize_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void FindAbsMaxKernel(const int n, const T* in, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ T shared_max_data[];
  if (gridDim.x > 1) {
    shared_max_data[tid] = T(0);
    for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
      T tmp = fabs(in[i]);
      if (tmp > shared_max_data[tid]) {
        shared_max_data[tid] = tmp;
      }
    }
  } else {
    if (bid < n) {
      shared_max_data[tid] = fabs(in[bid]);
    } else {
      shared_max_data[tid] = T(0);
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && shared_max_data[tid] < shared_max_data[tid + i]) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

float FindAbsMaxGpu(const platform::CUDADeviceContext& ctx, const float* array,
                    int length) {
  float host_max;
  int kNumTheads = 1024;
  int gridDimx = (kNumTheads - 1 + length) / kNumTheads;
  gridDimx = (gridDimx > kNumTheads) ? kNumTheads : gridDimx;
  framework::Tensor t;
  float* device_max = t.mutable_data<float>(framework::make_ddim({gridDimx}),
                                            platform::CUDAPlace());
  FindAbsMaxKernel<float><<<gridDimx, kNumTheads, kNumTheads * sizeof(float),
                            ctx.stream()>>>(length, array, device_max);
  FindAbsMaxKernel<
      float><<<1, kNumTheads, kNumTheads * sizeof(float), ctx.stream()>>>(
      gridDimx, device_max, device_max);
  PADDLE_ENFORCE_EQ(
      hipMemcpy(&host_max, device_max, sizeof(float), hipMemcpyDeviceToHost),
      hipSuccess, "hipMemcpy failed");
  return host_max;
}

template <typename T>
__global__ void ApplySaturateKernel(const int n, const T* in, T* out,
                                    int* num_saturate, const T min,
                                    const T max) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ int shared_count[];
  shared_count[tid] = 0;
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    if (in[i] > max) {
      out[i] = max;
      shared_count[tid] += 1;
    } else if (in[i] < min) {
      out[i] = min;
      shared_count[tid] += 1;
    } else {
      out[i] = in[i];
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i) {
      shared_count[tid] += shared_count[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    num_saturate[blockIdx.x] = shared_count[0];
  }
}

template <typename T>
__global__ void ReduceKernel(const int n, const T* in, T* out) {
  int tid = threadIdx.x;
  extern __shared__ T shared_sum[];
  if (tid < n) {
    shared_sum[tid] = in[tid];
  } else {
    shared_sum[tid] = T(0);
  }
  __syncthreads();
  // blockDim.x must >= n
  for (int i = (n + 1) / 2; i > 0; i >>= 1) {
    if (tid < i) {
      shared_sum[tid] += shared_sum[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[0] = shared_sum[0];
  }
}

template <typename T>
int ApplySaturateGpu(const platform::CUDADeviceContext& ctx, const int n,
                     const T* in, T* out, const T min, const T max) {
  int host_num_saturate;
  int kNumTheads = 1024;
  int gridDimx = (n + kNumTheads - 1) / kNumTheads;
  gridDimx = (gridDimx > kNumTheads) ? kNumTheads : gridDimx;
  framework::Tensor t;
  int* device_num_saturate = t.mutable_data<int>(
      framework::make_ddim({gridDimx}), platform::CUDAPlace());
  ApplySaturateKernel<
      T><<<gridDimx, kNumTheads, kNumTheads * sizeof(T), ctx.stream()>>>(
      n, in, out, device_num_saturate, min, max);
  ReduceKernel<int><<<1, kNumTheads, kNumTheads * sizeof(T), ctx.stream()>>>(
      gridDimx, device_num_saturate, device_num_saturate);
  PADDLE_ENFORCE_EQ(hipSuccess,
                    hipMemcpy(&host_num_saturate, device_num_saturate,
                               sizeof(int), hipMemcpyDeviceToHost),
                    "hipMemcpy failed");
  return host_num_saturate;
}

template <typename DeviceContext, typename T>
class FakeQuantizeCUDAKernel : public framework::OpKernel<T> {
 public:
  T FindRangeAbsMax(const platform::CUDADeviceContext& ctx,
                    framework::Tensor* scale_list, framework::Tensor* out_scale,
                    const T& cur_scale, int window_size,
                    int current_iter) const {
    T* sl = scale_list->mutable_data<T>(platform::CPUPlace());
    T remove_tmp = sl[current_iter];
    sl[current_iter] = cur_scale;
    T& max_scale = out_scale->mutable_data<T>(platform::CPUPlace())[0];
    if (max_scale < cur_scale) {
      max_scale = cur_scale;
    } else if (fabs(remove_tmp - max_scale) < 1e-6) {
      int size = (current_iter > window_size) ? window_size : current_iter;
      max_scale = T(FindAbsMaxGpu(ctx, scale_list->data<float>(), size));
    }
    return max_scale;
  }

  T FindMovingAverageAbsMmax(framework::Tensor* in_scale,
                             framework::Tensor* out_scale,
                             const T& cur_scale) const {
    T* ins = in_scale->mutable_data<T>(platform::CPUPlace());
    T* outs = out_scale->mutable_data<T>(platform::CPUPlace());
    outs[0] = 0.9 * cur_scale + 0.1 * ins[0];
    return T(outs[0]);
  }

  virtual void Compute(const framework::ExecutionContext& context) const {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto& device_ctx = context.cuda_device_context();
    auto* tensor = context.Output<framework::Tensor>("Out");
    auto* in = context.Input<framework::Tensor>("X");
    const bool is_test = context.Attr<bool>("is_test");
    tensor->mutable_data<T>(in->place());
    context.Output<framework::Tensor>("OutMovingScale")
        ->mutable_data<T>(
            context.Input<framework::Tensor>("InMovingScale")->place());
    auto quantize_type =
        static_cast<std::string>(context.Attr<std::string>("quantize_type"));
    if (quantize_type == std::string("range_abs_max")) {
      context.Output<framework::Tensor>("OutScales")
          ->mutable_data<T>(
              context.Input<framework::Tensor>("InScales")->place());
      context.Output<framework::Tensor>("OutCurrentIter")
          ->mutable_data<T>(
              context.Input<framework::Tensor>("InCurrentIter")->place());
    }

    T scale = T(1);
    int window_size = context.Attr<int>("window_size");
    T bin_cnt = (T)((1 << (context.Attr<int>("bit_length") - 1)) - 1);
    if (quantize_type == std::string("abs_max")) {
      auto* saving_scale = context.Output<framework::Tensor>("OutMovingScale");
      scale = (T)FindAbsMaxGpu(device_ctx, in->data<float>(), in->numel());
      saving_scale->mutable_data<T>(platform::CPUPlace())[0] = scale;

      auto& device_ctx = context.template device_context<DeviceContext>();
      auto* scale_list = context.Output<framework::Tensor>("OutScales");
      math::SetConstant<DeviceContext, T> scalar;
      scale_list->mutable_data<T>(context.GetPlace());
      scalar(device_ctx, scale_list, static_cast<T>(0));
      auto* iter = context.Output<framework::Tensor>("OutCurrentIter");
      iter->mutable_data<T>(context.GetPlace());
      scalar(device_ctx, iter, static_cast<T>(0));
    } else if (quantize_type == std::string("range_abs_max")) {
      auto* moving_scale = const_cast<framework::Tensor*>(
          context.Input<framework::Tensor>("InMovingScale"));
      if (is_test) {
        scale = moving_scale->mutable_data<T>(platform::CPUPlace())[0];
      } else {
        auto* it = const_cast<framework::Tensor*>(
            context.Input<framework::Tensor>("InCurrentIter"));
        auto* iter = context.Output<framework::Tensor>("OutCurrentIter");
        int* last_iter = it->mutable_data<int>(platform::CPUPlace());
        int* current_iter = iter->mutable_data<int>(platform::CPUPlace());
        auto* scale_list = context.Output<framework::Tensor>("OutScales");
        auto* saving_scale =
            context.Output<framework::Tensor>("OutMovingScale");
        scale = (T)FindAbsMaxGpu(device_ctx, in->data<float>(), in->numel());
        scale = FindRangeAbsMax(device_ctx, scale_list, saving_scale, scale,
                                window_size, current_iter[0]);
        (*current_iter) = (*last_iter) + 1;
      }
    } else if (quantize_type == std::string("moving_average_abs_max")) {
      auto* moving_scale = const_cast<framework::Tensor*>(
          context.Input<framework::Tensor>("InMovingScale"));
      if (is_test) {
        scale = moving_scale->mutable_data<T>(platform::CPUPlace())[0];
      } else {
        scale = (T)FindAbsMaxGpu(device_ctx, in->data<float>(), in->numel());
        auto* saving_scale =
            context.Output<framework::Tensor>("OutMovingScale");
        scale = FindMovingAverageAbsMmax(
            const_cast<framework::Tensor*>(moving_scale), saving_scale, scale);
      }
    }

    ApplySaturateGpu<T>(device_ctx, in->numel(), in->data<T>(),
                        tensor->mutable_data<T>(in->place()), -scale, scale);
    scale = bin_cnt / scale;

    auto& dev =
        *context.template device_context<DeviceContext>().eigen_device();
    auto eigen_out = framework::EigenVector<T>::Flatten(*tensor);
    auto eigen_in = framework::EigenVector<T>::Flatten(*tensor);
    eigen_out.device(dev) = (scale * eigen_in).round();
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(fake_quantize,
                        paddle::operators::FakeQuantizeCUDAKernel<
                            paddle::platform::CUDADeviceContext, float>,
                        paddle::operators::FakeQuantizeCUDAKernel<
                            paddle::platform::CUDADeviceContext, double>);
