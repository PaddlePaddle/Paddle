/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fake_quantize_op.h"
#include "paddle/fluid/operators/fake_quantize_op.cu.h"

namespace ops = paddle::operators;
using CUDA = phi::GPUContext;
using float16 = paddle::platform::float16;
REGISTER_OP_CUDA_KERNEL(fake_quantize_abs_max,
                        ops::FakeQuantizeAbsMaxKernel<CUDA, float>,
                        ops::FakeQuantizeAbsMaxKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_dequantize_abs_max,
                        ops::FakeQuantizeDequantizeAbsMaxKernel<CUDA, float>,
                        ops::FakeQuantizeDequantizeAbsMaxKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(
    fake_channel_wise_quantize_abs_max,
    ops::FakeChannelWiseQuantizeAbsMaxKernel<CUDA, float>,
    ops::FakeChannelWiseQuantizeAbsMaxKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_range_abs_max,
                        ops::FakeQuantizeRangeAbsMaxKernel<CUDA, float>,
                        ops::FakeQuantizeRangeAbsMaxKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_moving_average_abs_max,
    ops::FakeQuantizeMovingAverageAbsMaxKernel<CUDA, float>,
    ops::FakeQuantizeMovingAverageAbsMaxKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(moving_average_abs_max_scale,
                        ops::MovingAverageAbsMaxScaleKernel<CUDA, float>,
                        ops::MovingAverageAbsMaxScaleKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_dequantize_moving_average_abs_max,
    ops::FakeQuantizeDequantizeMovingAverageAbsMaxKernel<CUDA, float>,
    ops::FakeQuantizeDequantizeMovingAverageAbsMaxKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(stright_throuth_estimator_grad,
                        ops::StrightThroughEstimatorGradKernel<CUDA, float>,
                        ops::StrightThroughEstimatorGradKernel<CUDA, float16>);
REGISTER_OP_CUDA_KERNEL(
    fake_channel_wise_quantize_dequantize_abs_max,
    ops::FakeChannelWiseQuantizeDequantizeAbsMaxKernel<CUDA, float>);
