#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/fake_quantize_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void FindAbsMaxKernel(const T* in, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ T shared_max_data[];
  if (gridDim.x > 1) {
    shared_max_data[tid] = T(0);
    for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
      T tmp = fabs(in[i]);
      if (tmp > shared_max_data[tid]) {
        shared_max_data[tid] = tmp;
      }
    }
  } else {
    if (bid < n) {
      shared_max_data[tid] = fabs(in[bid]);
    } else {
      shared_max_data[tid] = T(0);
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i && (shared_max_data[tid] < shared_max_data[tid + i])) {
      shared_max_data[tid] = shared_max_data[tid + i];
    }
    __syncthreads();
  }
  if (tid == 0) {
    out[blockIdx.x] = shared_max_data[0];
  }
}

template <typename T>
struct FindAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx, const T* in,
                  const int num, T* out) {
    int block = 1024;
    int grid = (block - 1 + num) / block;
    grid = (grid > block) ? block : grid;

    framework::Tensor max;
    T* max_data =
        max.mutable_data<T>(framework::make_ddim({grid}), ctx.GetPlace());
    FindAbsMaxKernel<T><<<grid, block, 1024 * sizeof(T), ctx.stream()>>>(
        in, num, max_data);
    FindAbsMaxKernel<T><<<1, block, 1024 * sizeof(T), ctx.stream()>>>(
        max_data, grid, out);
  }
};

template struct FindAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
__global__ void ClipAndQuantKernel(const T* in, const T* scale,
                                   const int bin_cnt, const int n, T* out) {
  int bid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  T s = scale[0];
  for (int i = bid; i < n; i += blockDim.x * gridDim.x) {
    T x = in[bid];
    T v = x > s ? s : x;
    v = v < -s ? -s : v;
    v = bin_cnt / s * v;
    out[bid] = round(v);
  }
}

template <typename T>
__global__ void FindRangeAbsMaxAndFillArray(const T* cur_scale,
                                            const T* last_scale,
                                            const int64_t* iter,
                                            const int window_size, T* scale_arr,
                                            T* out_scale, int* need_find_max,
                                            int* out_size) {
  int it = iter[0];
  int idx = it % window_size;
  T removed = scale_arr[idx];
  T cur = cur_scale[0];
  scale_arr[idx] = cur;
  T max = last_scale[0];
  out_scale[0] = max < cur ? cur : max;
  if (fabs(removed - max) < 1e-6) {
    need_find_max[0] = 1;
    out_size[0] = it > window_size ? window_size : it;
  } else {
    need_find_max[0] = 0;
  }
}

template <typename T>
struct FindRangeAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& cur_scale,
                  const framework::Tensor& last_scale,
                  const framework::Tensor& iter, const int window_size,
                  framework::Tensor* scales_arr, framework::Tensor* out_scale) {
    const auto gpu_place = boost::get<platform::CUDAPlace>(ctx.GetPlace());

    T* scale_arr = scales_arr->mutable_data<T>(gpu_place);
    T* out_scale_data = out_scale->mutable_data<T>(gpu_place);

    framework::Tensor need_find_max, out_size;
    int* find_max = need_find_max.mutable_data<int>(gpu_place);
    int* out_size_data = out_size.mutable_data<int>(gpu_place);

    FindRangeAbsMaxAndFillArray<T><<<1, 1, 0, ctx.stream()>>>(
        cur_scale.data<T>(), last_scale.data<T>(), iter.data<int64_t>(),
        window_size, scale_arr, out_scale_data, find_max, out_size_data);

    int g_find_max;
    memory::Copy(platform::CPUPlace(), &g_find_max, gpu_place, find_max,
                 sizeof(int), 0);
    if (g_find_max) {
      int len;
      memory::Copy(platform::CPUPlace(), &len, gpu_place, out_size_data,
                   sizeof(int), 0);
      FindAbsMaxFunctor<platform::CUDADeviceContext, T>()(ctx, scale_arr, len,
                                                          out_scale_data);
    }
  }
};

template struct FindRangeAbsMaxFunctor<platform::CUDADeviceContext, float>;

template <typename T>
struct FindMovingAverageAbsMaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in_accum,
                  const framework::Tensor& in_state, const T* cur_scale,
                  const float rate, framework::Tensor* out_state,
                  framework::Tensor* out_accum, framework::Tensor* out_scale) {
    const auto gpu_place = boost::get<platform::CUDAPlace>(ctx.GetPlace());

    T accum;
    memory::Copy(platform::CPUPlace(), &accum, gpu_place, in_accum.data<T>(),
                 sizeof(T), 0);
    T state;
    memory::Copy(platform::CPUPlace(), &state, gpu_place, in_state.data<T>(),
                 sizeof(T), 0);
    T scale;
    memory::Copy(platform::CPUPlace(), &scale, gpu_place, cur_scale, sizeof(T),
                 0);

    state = rate * state + 1;
    accum = rate * accum + scale;
    scale = accum / state;

    memory::Copy(gpu_place, out_accum->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &accum, sizeof(T), 0);
    memory::Copy(gpu_place, out_state->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &state, sizeof(T), 0);
    memory::Copy(gpu_place, out_scale->mutable_data<T>(gpu_place),
                 platform::CPUPlace(), &scale, sizeof(T), 0);
  }
};

template struct FindMovingAverageAbsMaxFunctor<platform::CUDADeviceContext,
                                               float>;

template <typename T>
struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& in, const framework::Tensor& scale,
                  const int bin_cnt, framework::Tensor* out) {
    int num = in.numel();
    int block = 1024;
    int grid = (block - 1 + num) / block;

    const T* in_data = in.data<T>();
    const T* scale_data = scale.data<T>();
    T* out_data = out->mutable_data<T>(ctx.GetPlace());

    ClipAndQuantKernel<T><<<grid, block, 0, ctx.stream()>>>(
        in_data, scale_data, bin_cnt, num, out_data);
  }
};

template struct ClipAndFakeQuantFunctor<platform::CUDADeviceContext, float>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(fake_quantize_abs_max,
                        ops::FakeQuantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_channel_wise_quantize_abs_max,
                        ops::FakeChannelWiseQuantizeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(fake_quantize_range_abs_max,
                        ops::FakeQuantizeRangeAbsMaxKernel<CUDA, float>);
REGISTER_OP_CUDA_KERNEL(
    fake_quantize_moving_average_abs_max,
    ops::FakeQuantizeMovingAverageAbsMaxKernel<CUDA, float>);
