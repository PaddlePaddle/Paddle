#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fake_dequantize_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void KeDequantize(const T* in, const T* scale, T max_range, int num,
                             T* out) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < num) {
    out[idx] = in[idx] * scale[0] / max_range;
  }
}

template <typename T>
struct DequantizeFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor* in, const framework::Tensor* scale,
                  T max_range, framework::Tensor* out) {
    const T* in_data = in->data<T>();
    const T* scale_factor = scale->data<T>();
    T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());

    int num = in->numel();
    int block = 512;
    int grid = (num + block - 1) / block;

    KeDequantize<T><<<grid, block, 0, dev_ctx.stream()>>>(
        in_data, scale_factor, max_range, num, out_data);
  }
};

template <typename T>
__global__ void DequantizeOneScale(const T* in, const T* scale, T max_range,
                                   int num, int channel, T* out) {
  int tid = threadIdx.x;
  int channel_size = num / channel;
  const T* in_c = in + blockIdx.x * channel_size;
  T* out_c = out + blockIdx.x * channel_size;
  for (int i = tid; i < channel_size; i += blockDim.x) {
    out_c[i] = in_c[i] * scale[blockIdx.x] / max_range;
  }
}

template <typename T>
__global__ void DequantizeTwoScale(const T* in, const T* scale_one,
                                   const T* scale_two, T max_range, int num,
                                   int batch_size, int channel, T* out) {
  int tid = threadIdx.x;
  int channel_size = num / (batch_size * channel);
  int scale_index = blockIdx.x % channel;
  const T* in_c = in + blockIdx.x * channel_size;
  T* out_c = out + blockIdx.x * channel_size;
  for (int i = tid; i < channel_size; i += blockDim.x) {
    out_c[i] = in_c[i] * scale_one[scale_index] * scale_two[0] / max_range;
  }
}

template <typename T>
struct ChannelDequantizeFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor* in, const framework::Tensor** scales,
                  const int scale_num, T max_range, framework::Tensor* out) {
    const T* in_data = in->data<T>();
    T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());
    if (scale_num == 1) {
      int num = in->numel();
      int channel = in->dims()[0];
      const T* scale_factor = scales[0]->data<T>();
      int block = 1024;
      int grid = channel;
      DequantizeOneScale<T><<<grid, block, 0, dev_ctx.stream()>>>(
          in_data, scale_factor, max_range, num, channel, out_data);
    } else if (scale_num == 2) {
      int num = in->numel();
      int batch_size = in->dims()[0];
      int channel = in->dims()[1];
      const T* scale_one = scales[0]->data<T>();
      const T* scale_two = scales[1]->data<T>();
      int block = 1024;
      int grid = batch_size * channel;
      DequantizeTwoScale<T><<<grid, block, 0, dev_ctx.stream()>>>(
          in_data, scale_one, scale_two, max_range, num, batch_size, channel,
          out_data);
    }
  }
};

template struct DequantizeFunctor<platform::CUDADeviceContext, float>;
template struct DequantizeFunctor<platform::CUDADeviceContext, double>;
template struct ChannelDequantizeFunctor<platform::CUDADeviceContext, float>;
template struct ChannelDequantizeFunctor<platform::CUDADeviceContext, double>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(fake_dequantize_max_abs,
                        ops::FakeDequantizeMaxAbsKernel<CUDA, float>,
                        ops::FakeDequantizeMaxAbsKernel<CUDA, double>);
REGISTER_OP_CUDA_KERNEL(
    fake_channel_wise_dequantize_max_abs,
    ops::FakeChannelWiseDequantizeMaxAbsKernel<CUDA, float>,
    ops::FakeChannelWiseDequantizeMaxAbsKernel<CUDA, double>);
