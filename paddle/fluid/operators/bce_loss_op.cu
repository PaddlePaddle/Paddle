#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <algorithm>
#include "paddle/fluid/operators/bce_loss_op.h"
#include "paddle/fluid/operators/math.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"
#include "paddle/fluid/platform/hostdevice.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void GPUBCELossForward(const T* x_data, const T* label_data,
                                  T* out_data, const int in_numel) {
  CUDA_KERNEL_LOOP(i, in_numel) {
    T x = x_data[i];
    T label = label_data[i];
    T one = static_cast<T>(1.);
    T neg_100 = static_cast<T>(-100.);

    PADDLE_ENFORCE(
        (x >= static_cast<T>(0)) && (x <= one),
        "Input is expected to be within the interval [0, 1], but recieved %f.",
        x);

    T term1 = max(real_log(x), neg_100);
    T term2 = max(real_log(one - x), neg_100);

    out_data[i] = ((label - one) * term2) - (label * term1);
  }
}

template <typename T>
__global__ void GPUBCELossBackward(const T* x_data, const T* label_data,
                                   const T* dout_data, T* dx_data,
                                   const int in_numel) {
  CUDA_KERNEL_LOOP(i, in_numel) {
    T x = x_data[i];
    T label = label_data[i];
    T dout = dout_data[i];
    T one = static_cast<T>(1.);
    T eps = static_cast<T>(1e-12);

    T term1 = max((one - x) * x, eps);

    dx_data[i] = dout * (x - label) / term1;
  }
}

template <typename DeviceContext, typename T>
class BCELossCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* labels = ctx.Input<Tensor>("Label");
    auto* out = ctx.Output<Tensor>("Out");

    const auto* x_data = x->data<T>();
    auto* out_data = out->mutable_data<T>(ctx.GetPlace());
    auto x_numel = x->numel();

    auto& dev_ctx = ctx.cuda_device_context();
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(dev_ctx, x_numel);

    GPUBCELossForward<T><<<config.block_per_grid, config.thread_per_block, 0,
                           dev_ctx.stream()>>>(x_data, labels->data<T>(),
                                               out_data, x_numel);
  }
};

template <typename DeviceContext, typename T>
class BCELossGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* labels = ctx.Input<Tensor>("Label");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dx = ctx.Output<Tensor>(framework::GradVarName("X"));

    int x_numel = x->numel();
    auto* dx_data = dx->mutable_data<T>(ctx.GetPlace());

    auto& dev_ctx = ctx.cuda_device_context();
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(dev_ctx, x_numel);

    GPUBCELossBackward<T><<<config.block_per_grid, config.thread_per_block, 0,
                            dev_ctx.stream()>>>(
        x->data<T>(), labels->data<T>(), dout->data<T>(), dx_data, x_numel);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    bce_loss,
    ops::BCELossCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::BCELossCUDAKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    bce_loss_grad,
    ops::BCELossGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::BCELossGradCUDAKernel<paddle::platform::CUDADeviceContext, double>);
