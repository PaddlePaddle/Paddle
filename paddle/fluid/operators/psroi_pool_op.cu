#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/psroi_pool_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__global__ void GPUPSROIPoolForward(
    const int nthreads, const T* input_data, const T* input_rois,
    const float spatial_scale, const int input_channels, const int height,
    const int width, const int output_channels, const int pooled_height,
    const int pooled_width, const int* rois_batch_id_data, T* output_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    // The output is in order (n, c, ph, pw)
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % output_channels;
    int n = i / pooled_width / pooled_height / output_channels;

    // set roi_batch_id
    int roi_batch_id = rois_batch_id_data[n];

    // [start, end) interval for spatial sampling
    const T* offset_input_rois = input_rois + n * 4;
    T roi_start_w = static_cast<T>(round(offset_input_rois[0])) * spatial_scale;
    T roi_start_h = static_cast<T>(round(offset_input_rois[1])) * spatial_scale;
    T roi_end_w =
        static_cast<T>(round(offset_input_rois[2]) + 1.) * spatial_scale;
    T roi_end_h =
        static_cast<T>(round(offset_input_rois[3]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    T roi_height = max(roi_end_h - roi_start_h, (T)0.1);  // avoid 0
    T roi_width = max(roi_end_w - roi_start_w, (T)0.1);

    // Compute w and h at input feature map
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    int hstart = floor(bin_size_h * static_cast<T>(ph) + roi_start_h);
    int wstart = floor(bin_size_w * static_cast<T>(pw) + roi_start_w);
    int hend = ceil(bin_size_h * static_cast<T>(ph + 1) + roi_start_h);
    int wend = ceil(bin_size_w * static_cast<T>(pw + 1) + roi_start_w);

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int input_channel = (c * pooled_height + ph) * pooled_width + pw;
    const T* offset_input_data =
        input_data +
        (roi_batch_id * input_channels + input_channel) * height * width;
    T outsum = 0;

    for (int ih = hstart; ih < hend; ++ih) {
      for (int iw = wstart; iw < wend; ++iw) {
        int input_index = ih * width + iw;
        outsum += offset_input_data[input_index];
      }
    }

    T bin_area = static_cast<T>((hend - hstart) * (wend - wstart));
    output_data[i] = is_empty ? 0. : outsum / bin_area;
  }
}

template <typename T>
__global__ void GPUPSROIPoolBackward(
    const int nthreads, const T* input_rois, const T* output_grad_data,
    const float spatial_scale, const int input_channels, const int height,
    const int width, const int output_channels, const int pooled_height,
    const int pooled_width, const int* rois_batch_id_data, T* input_grad_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    // The output is in order (n, c, ph, pw)
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % output_channels;
    int n = i / pooled_width / pooled_height / output_channels;

    // set roi_batch_id
    int roi_batch_id = rois_batch_id_data[n];
    int input_channel = (c * pooled_height + ph) * pooled_width + pw;
    int input_offset =
        (roi_batch_id * input_channels + input_channel) * height * width;
    T* offset_input_grad_data = input_grad_data + input_offset;

    // [start, end) interval for spatial sampling
    const T* offset_input_rois = input_rois + n * 4;
    T roi_start_w = static_cast<T>(round(offset_input_rois[0])) * spatial_scale;
    T roi_start_h = static_cast<T>(round(offset_input_rois[1])) * spatial_scale;
    T roi_end_w =
        static_cast<T>(round(offset_input_rois[2]) + 1.) * spatial_scale;
    T roi_end_h =
        static_cast<T>(round(offset_input_rois[3]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    T roi_height = max(roi_end_h - roi_start_h, (T)0.1);  // avoid 0
    T roi_width = max(roi_end_w - roi_start_w, (T)0.1);

    // Compute w and h at input feature map
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    int hstart = floor(bin_size_h * static_cast<T>(ph) + roi_start_h);
    int wstart = floor(bin_size_w * static_cast<T>(pw) + roi_start_w);
    int hend = ceil(bin_size_h * static_cast<T>(ph + 1) + roi_start_h);
    int wend = ceil(bin_size_w * static_cast<T>(pw + 1) + roi_start_w);

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Accumulate diff_val into input data
    T bin_area = static_cast<T>((hend - hstart) * (wend - wstart));
    T diff_val = is_empty ? 0. : output_grad_data[i] / bin_area;
    for (int ih = hstart; ih < hend; ++ih) {
      for (int iw = wstart; iw < wend; ++iw) {
        int input_index = ih * width + iw;
        platform::CudaAtomicAdd(offset_input_grad_data + input_index, diff_val);
      }
    }
  }
}

template <typename Place, typename T>
class GPUPSROIPoolOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");
    auto* out = ctx.Output<Tensor>("Out");

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto output_channels = ctx.Attr<int>("output_channels");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    auto in_dims = in->dims();
    int batch_size = in_dims[0];
    int input_channels = in_dims[1];
    int height = in_dims[2];
    int width = in_dims[3];

    PADDLE_ENFORCE_EQ(input_channels,
                      output_channels * pooled_height * pooled_width,
                      "the channels of input X should equal the product of "
                      "output_channels x pooled_height x pooled_width");

    int rois_num = rois->dims()[0];
    if (rois_num == 0) return;

    auto rois_lod = rois->lod().back();
    int rois_batch_size = rois_lod.size() - 1;
    PADDLE_ENFORCE_EQ(
        rois_batch_size, batch_size,
        "The rois_batch_size and input(X) batch_size must be the same.");
    int rois_num_with_lod = rois_lod[rois_batch_size];
    PADDLE_ENFORCE_EQ(rois_num, rois_num_with_lod,
                      "The rois_num from input and lod must be the same.");

    // set rois batch id
    framework::Tensor rois_batch_id_list;
    rois_batch_id_list.Resize({rois_num});
    int* rois_batch_id_data =
        rois_batch_id_list.mutable_data<int>(platform::CPUPlace());
    for (int n = 0; n < rois_batch_size; ++n) {
      for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
        rois_batch_id_data[i] = n;
      }
    }

    framework::Tensor rois_batch_id_list_gpu;
    framework::TensorCopy(rois_batch_id_list, ctx.GetPlace(),
                          ctx.device_context(), &rois_batch_id_list_gpu);

    int output_size = out->numel();
    int blocks = NumBlocks(output_size);
    int threads = kNumCUDAThreads;

    // call cuda kernel function
    GPUPSROIPoolForward<
        T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
        output_size, in->data<T>(), rois->data<T>(), spatial_scale,
        input_channels, height, width, output_channels, pooled_height,
        pooled_width, rois_batch_id_list_gpu.data<int>(),
        out->mutable_data<T>(ctx.GetPlace()));
  }
};

template <typename Place, typename T>
class GPUPSROIPoolGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");

    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto output_channels = ctx.Attr<int>("output_channels");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");

    int rois_num = rois->dims()[0];
    int input_channels = in->dims()[1];
    int height = in->dims()[2];
    int width = in->dims()[3];

    if (input_grad) {
      // set roi batch id
      framework::Tensor rois_batch_id_list;
      rois_batch_id_list.Resize({rois_num});
      int* rois_batch_id_data =
          rois_batch_id_list.mutable_data<int>(platform::CPUPlace());
      auto rois_lod = rois->lod().back();
      int rois_batch_size = rois_lod.size() - 1;
      for (int n = 0; n < rois_batch_size; ++n) {
        for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
          rois_batch_id_data[i] = n;
        }
      }

      framework::Tensor rois_batch_id_list_gpu;
      framework::TensorCopy(rois_batch_id_list, ctx.GetPlace(),
                            ctx.device_context(), &rois_batch_id_list_gpu);

      input_grad->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<Place, T> set_zero;
      set_zero(ctx.cuda_device_context(), input_grad, static_cast<T>(0));

      int output_grad_size = output_grad->numel();
      int blocks = NumBlocks(output_grad_size);
      int threads = kNumCUDAThreads;

      if (output_grad_size > 0) {
        GPUPSROIPoolBackward<
            T><<<blocks, threads, 0, ctx.cuda_device_context().stream()>>>(
            output_grad_size, rois->data<T>(), output_grad->data<T>(),
            spatial_scale, input_channels, height, width, output_channels,
            pooled_height, pooled_width, rois_batch_id_list_gpu.data<int>(),
            input_grad->mutable_data<T>(ctx.GetPlace()));
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    psroi_pool,
    ops::GPUPSROIPoolOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUPSROIPoolOpKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    psroi_pool_grad,
    ops::GPUPSROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUPSROIPoolGradOpKernel<paddle::platform::CUDADeviceContext, double>);
