#include "hip/hip_runtime.h"
/* Copyright (c) 2010 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/affine_grid_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void LinspaceKernel(T start, T step, int64_t size, T* out) {
  CUDA_KERNEL_LOOP(index, size) { out[index] = start + step * index; }
}

template <typename T>
struct Linspace<paddle::platform::CUDADeviceContext, T> {
  void operator()(T start, T end, int count, bool align_corners,
                  framework::Tensor* numbers,
                  const framework::ExecutionContext& ctx) {
    T* number_data = numbers->mutable_data<T>({count}, ctx.GetPlace());
    T slice = (end - start) / (T)(count - 1);
    if (!align_corners) {
      slice = (end - start) / (T)count;
      start *= (T)(count - 1) / (T)count;
    }
    auto stream = ctx.cuda_device_context().stream();
    int block = 512;
    int grid = (count + block - 1) / block;
    LinspaceKernel<T><<<grid, block, 0, stream>>>(start, slice, count,
                                                  number_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    affine_grid,
    ops::AffineGridOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::AffineGridOpKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    affine_grid_grad,
    ops::AffineGridGradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::AffineGridGradOpKernel<paddle::platform::CUDADeviceContext, double>);
