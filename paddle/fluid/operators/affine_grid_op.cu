#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/affine_grid_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void LinspaceKernel(T start, T step, int64_t size, T* out) {
  CUDA_KERNEL_LOOP(index, size) { out[index] = start + step * index; }
}

template <typename T>
struct Linspace<paddle::platform::CUDADeviceContext, T> {
  void operator()(T start, T end, int count, bool align_corners,
                  framework::Tensor* numbers,
                  const framework::ExecutionContext& ctx) {
    T* number_data = numbers->mutable_data<T>({count}, ctx.GetPlace());
    T slice = (end - start) / (T)(count - 1);
    if (!align_corners) {
      slice = (end - start) / (T)count;
      start *= (T)(count - 1) / (T)count;
    }
    auto stream = ctx.cuda_device_context().stream();
    int block = 512;
    int grid = (count + block - 1) / block;
    LinspaceKernel<T><<<grid, block, 0, stream>>>(start, slice, count,
                                                  number_data);
  }
};

template <typename T>
__global__ void affine_grid_kernel(const int count, int n, int out_h, int out_w,
                                   T h_start, T w_start, T h_step, T w_step,
                                   const T* theta,  // N, 2, 3
                                   T* output) {
  CUDA_KERNEL_LOOP(index, count) {
    int w = index % out_w;
    int h = (index / out_w) % out_h;
    int n = index / (out_w * out_h);

    T h_coor = h_step * static_cast<T>(h) + static_cast<T>(h_start);
    T w_coor = w_step * static_cast<T>(w) + static_cast<T>(w_start);

    int theta_offset = n * 6;  // 2 * 3;
    // affine from (h_coor, w_coor) to (x, y)
    output[index * 2] = theta[theta_offset] * w_coor +
                        theta[theta_offset + 1] * h_coor +
                        theta[theta_offset + 2];
    output[index * 2 + 1] = theta[theta_offset + 3] * w_coor +
                            theta[theta_offset + 4] * h_coor +
                            theta[theta_offset + 5];
  }
}

template <typename T>
__global__ void affine_grid_grad_kernel(const int count, int n, int out_h,
                                        int out_w, T h_start, T w_start,
                                        T h_step, T w_step,
                                        const T* out_grad,  // N, H, W, 2
                                        T* theta_grad) {    // N, 2, 3
  CUDA_KERNEL_LOOP(index, count) {
    int w = index % out_w;
    int h = (index / out_w) % out_h;
    int n = index / (out_w * out_h);
    T h_coor = h_step * static_cast<T>(h) + static_cast<T>(h_start);
    T w_coor = w_step * static_cast<T>(w) + static_cast<T>(w_start);

    int theta_offset = n * 6;  // 2 * 3;
    T out_grad_x = out_grad[index * 2];
    platform::CudaAtomicAdd(theta_grad + theta_offset, out_grad_x * w_coor);
    platform::CudaAtomicAdd(theta_grad + theta_offset + 1, out_grad_x * h_coor);
    platform::CudaAtomicAdd(theta_grad + theta_offset + 2, out_grad_x);

    T out_grad_y = out_grad[index * 2 + 1];
    platform::CudaAtomicAdd(theta_grad + theta_offset + 3, out_grad_y * w_coor);
    platform::CudaAtomicAdd(theta_grad + theta_offset + 4, out_grad_y * h_coor);
    platform::CudaAtomicAdd(theta_grad + theta_offset + 5, out_grad_y);
  }
}

template <typename T>
class AffineGridOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* theta = ctx.Input<Tensor>("Theta");
    int n = theta->dims()[0];
    auto size_attr = ctx.Attr<std::vector<int>>("output_shape");
    auto align_corners = ctx.Attr<bool>("align_corners");
    int h = 0;
    int w = 0;
    if (size_attr.size() == 0) {
      auto* output_shape = ctx.Input<Tensor>("OutputShape");
      Tensor h_sizes;
      framework::TensorCopy(*output_shape, platform::CPUPlace(), &h_sizes);
      const int* h_size_data = h_sizes.data<int>();
      h = h_size_data[2];
      w = h_size_data[3];
    } else {
      h = size_attr[2];
      w = size_attr[3];
    }
    auto* output = ctx.Output<Tensor>("Output");
    T* out_data = output->mutable_data<T>({n, h, w, 2}, ctx.GetPlace());

    T h_step;
    T w_step;
    T h_start = -1;
    T w_start = -1;
    if (align_corners) {
      h_step = static_cast<T>(2) / static_cast<T>(h - 1);
      w_step = static_cast<T>(2) / static_cast<T>(w - 1);
    } else {
      h_step = static_cast<T>(2) / static_cast<T>(h);
      w_step = static_cast<T>(2) / static_cast<T>(w);

      h_start *= static_cast<T>(h - 1) / static_cast<T>(h);
      w_start *= static_cast<T>(w - 1) / static_cast<T>(w);
    }

    const int count = n * h * w;
    int block = 512;
    int grid = (count + block - 1) / block;
    auto cu_stream = ctx.cuda_device_context().stream();
    affine_grid_kernel<<<grid, block, 0, cu_stream>>>(
        count, n, h, w, h_start, w_start, h_step, w_step,
        theta->data<T>(),  // N, 2, 3
        out_data);
  }
};

template <typename T>
class AffineGridGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto theta_grad = ctx.Output<Tensor>(framework::GradVarName("Theta"));
    int n = output_grad->dims()[0];
    auto size_attr = ctx.Attr<std::vector<int>>("output_shape");
    auto align_corners = ctx.Attr<bool>("align_corners");
    int h = 0;
    int w = 0;
    if (size_attr.size() == 0) {
      auto* output_shape = ctx.Input<Tensor>("OutputShape");
      Tensor h_sizes;
      framework::TensorCopy(*output_shape, platform::CPUPlace(), &h_sizes);
      const int* h_size_data = h_sizes.data<int>();
      h = h_size_data[2];
      w = h_size_data[3];
    } else {
      h = size_attr[2];
      w = size_attr[3];
    }
    T* theta_grad_data = theta_grad->mutable_data<T>({n, 2, 3}, ctx.GetPlace());
    math::SetConstant<paddle::platform::CUDADeviceContext, T>()(
        ctx.cuda_device_context(), theta_grad, static_cast<T>(0));

    T h_step;
    T w_step;
    T h_start = -1;
    T w_start = -1;
    if (align_corners) {
      h_step = static_cast<T>(2) / static_cast<T>(h - 1);
      w_step = static_cast<T>(2) / static_cast<T>(w - 1);
    } else {
      h_step = static_cast<T>(2) / static_cast<T>(h);
      w_step = static_cast<T>(2) / static_cast<T>(w);

      h_start *= static_cast<T>(h - 1) / static_cast<T>(h);
      w_start *= static_cast<T>(w - 1) / static_cast<T>(w);
    }
    const int count = n * h * w;
    VLOG(3) << "count: " << count << "; h_step: " << h_step
            << "; w_step: " << w_step << "; h_start: " << h_start
            << "; w_start: " << w_start;
    int block = 512;
    int grid = (count + block - 1) / block;
    auto cu_stream = ctx.cuda_device_context().stream();
    affine_grid_grad_kernel<<<grid, block, 0, cu_stream>>>(
        count, n, h, w, h_start, w_start, h_step, w_step,
        output_grad->data<T>(), theta_grad_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(affine_grid, ops::AffineGridOpCUDAKernel<float>,
                        ops::AffineGridOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(affine_grid_grad,
                        ops::AffineGridGradOpCUDAKernel<float>,
                        ops::AffineGridGradOpCUDAKernel<double>);
