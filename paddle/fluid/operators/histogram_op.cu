#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/histogram_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"
#include "paddle/fluid/platform/hostdevice.h"

namespace paddle {
namespace operators {

using IndexType = int64_t;
using Tensor = framework::Tensor;
using platform::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename IndexType>
__device__ static IndexType GetBin(T input_value, T min_value, T max_value,
                                   int64_t nbins) {
  IndexType bin = static_cast<int>((input_value - min_value) * nbins /
                                   (max_value - min_value));
  IndexType output_index = bin < nbins - 1 ? bin : nbins - 1;
  return output_index;
}

template <typename T, typename IndexType>
__global__ void KernelHistogram(const T* input, const int total_elements,
                                const int64_t nbins, const T min_value,
                                const T max_value, int64_t* output) {
  extern __shared__ int64_t buf_hist[];
  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    buf_hist[i] = 0;
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(input_index, total_elements) {
    // const IndexType input_index = threadIdx.x + blockIdx.x * blockDim.x;
    const auto input_value = input[input_index];
    if (input_value >= min_value && input_value <= max_value) {
      const IndexType output_index =
          GetBin<T, IndexType>(input_value, min_value, max_value, nbins);
      paddle::platform::CudaAtomicAdd(&buf_hist[output_index], 1);
    }
  }
  __syncthreads();

  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    paddle::platform::CudaAtomicAdd(&output[i], buf_hist[i]);
  }
}

template <typename DeviceContext, typename T>
class HistogramCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));

    const Tensor* input = context.Input<framework::Tensor>("X");
    Tensor* output = context.Output<framework::Tensor>("Out");
    auto& nbins = context.Attr<int64_t>("bins");
    auto& minval = context.Attr<int>("min");
    auto& maxval = context.Attr<int>("max");

    const T* input_data = input->data<T>();
    const int input_numel = input->numel();

    T output_min = static_cast<T>(minval);
    T output_max = static_cast<T>(maxval);

    if (output_min == output_max) {
      auto input_x = framework::EigenVector<T>::Flatten(*input);

      framework::Tensor input_min_t, input_max_t;
      auto* input_min_data =
          input_min_t.mutable_data<T>({1}, context.GetPlace());
      auto* input_max_data =
          input_max_t.mutable_data<T>({1}, context.GetPlace());
      auto input_min_scala = framework::EigenScalar<T>::From(input_min_t);
      auto input_max_scala = framework::EigenScalar<T>::From(input_max_t);

      auto* place =
          context.template device_context<DeviceContext>().eigen_device();
      input_min_scala.device(*place) = input_x.minimum();
      input_max_scala.device(*place) = input_x.maximum();

      Tensor input_min_cpu, input_max_cpu;
      TensorCopySync(input_min_t, platform::CPUPlace(), &input_min_cpu);
      TensorCopySync(input_max_t, platform::CPUPlace(), &input_max_cpu);

      output_min = input_min_cpu.data<T>()[0];
      output_max = input_max_cpu.data<T>()[0];
    }
    if (output_min == output_max) {
      output_min = output_min - 1;
      output_max = output_max + 1;
    }

    PADDLE_ENFORCE_EQ(
        (std::isinf(static_cast<float>(output_min)) ||
         std::isnan(static_cast<float>(output_max)) ||
         std::isinf(static_cast<float>(output_min)) ||
         std::isnan(static_cast<float>(output_max))),
        false, platform::errors::OutOfRange("range of min, max is not finite"));
    PADDLE_ENFORCE_GE(
        output_max, output_min,
        platform::errors::InvalidArgument(
            "max must be larger or equal to min. If min and max are both zero, "
            "the minimum and maximum values of the data are used. "
            "But received max is %d, min is %d",
            maxval, minval));

    int64_t* out_data = output->mutable_data<int64_t>(context.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, int64_t>()(
        context.template device_context<platform::CUDADeviceContext>(), output,
        static_cast<int64_t>(0));

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    KernelHistogram<
        T, IndexType><<<GET_BLOCKS(input_numel), PADDLE_CUDA_NUM_THREADS,
                        nbins * sizeof(int64_t), stream>>>(
        input_data, input_numel, nbins, output_min, output_max, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    histogram,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, double>);
