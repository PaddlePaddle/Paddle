#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/histogram_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"
#include "paddle/fluid/platform/hostdevice.h"

namespace paddle {
namespace operators {

using IndexType = int64_t;
using Tensor = framework::Tensor;
using platform::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename IndexType>
__device__ static IndexType GetBin(T bVal, T minvalue, T maxvalue,
                                   int64_t nbins) {
  IndexType bin =
      static_cast<int>((bVal - minvalue) * nbins / (maxvalue - minvalue));
  if (bin == nbins) bin -= 1;
  return bin;
}

template <typename T, typename IndexType>
__global__ void KernelHistogram(const T* input, const int totalElements,
                                const int64_t nbins, const T minvalue,
                                const T maxvalue, int64_t* output) {
  CUDA_KERNEL_LOOP(linearIndex, totalElements) {
    const IndexType inputIdx = threadIdx.x + blockIdx.x * blockDim.x;
    const auto inputVal = input[inputIdx];
    if (inputVal >= minvalue && inputVal <= maxvalue) {
      const IndexType bin =
          GetBin<T, IndexType>(inputVal, minvalue, maxvalue, nbins);
      const IndexType outputIdx = bin < nbins - 1 ? bin : nbins - 1;
      paddle::platform::CudaAtomicAdd(&output[outputIdx], 1);
    }
  }
}

template <typename DeviceContext, typename T>
class HistogramCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));

    const Tensor* input = context.Input<framework::Tensor>("X");
    Tensor* output = context.Output<framework::Tensor>("Out");
    auto& nbins = context.Attr<int64_t>("bins");
    auto& minval = context.Attr<int>("min");
    auto& maxval = context.Attr<int>("max");

    const T* input_data = input->data<T>();
    const int input_numel = input->numel();

    T output_min = static_cast<T>(minval);
    T output_max = static_cast<T>(maxval);

    if (output_min == output_max) {
      auto input_x = framework::EigenVector<T>::Flatten(*input);

      framework::Tensor input_min_t, input_max_t;
      auto* input_min_data =
          input_min_t.mutable_data<T>({1}, context.GetPlace());
      auto* input_max_data =
          input_max_t.mutable_data<T>({1}, context.GetPlace());
      auto input_min_scala = framework::EigenScalar<T>::From(input_min_t);
      auto input_max_scala = framework::EigenScalar<T>::From(input_max_t);

      auto* place =
          context.template device_context<DeviceContext>().eigen_device();
      input_min_scala.device(*place) = input_x.minimum();
      input_max_scala.device(*place) = input_x.maximum();

      Tensor input_min_cpu, input_max_cpu;
      TensorCopySync(input_min_t, platform::CPUPlace(), &input_min_cpu);
      TensorCopySync(input_max_t, platform::CPUPlace(), &input_max_cpu);

      output_min = input_min_cpu.data<T>()[0];
      output_max = input_max_cpu.data<T>()[0];
    }
    if (output_min == output_max) {
      output_min = output_min - 1;
      output_max = output_max + 1;
    }

    PADDLE_ENFORCE_EQ(
        (std::isinf(static_cast<float>(output_min)) ||
         std::isnan(static_cast<float>(output_max)) ||
         std::isinf(static_cast<float>(output_min)) ||
         std::isnan(static_cast<float>(output_max))),
        false, platform::errors::OutOfRange("range of min, max is not finite"));
    PADDLE_ENFORCE_GE(
        output_max, output_min,
        platform::errors::InvalidArgument(
            "max must be larger or equal to min. If min and max are both zero, "
            "the minimum and maximum values of the data are used. "
            "But received max is %d, min is %d",
            maxval, minval));

    int64_t* out_data = output->mutable_data<int64_t>(context.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, int64_t>()(
        context.template device_context<platform::CUDADeviceContext>(), output,
        static_cast<int64_t>(0));

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    KernelHistogram<T, IndexType><<<GET_BLOCKS(input_numel),
                                    PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        input_data, input_numel, nbins, output_min, output_max, out_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    histogram,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::HistogramCUDAKernel<paddle::platform::CUDADeviceContext, double>);
