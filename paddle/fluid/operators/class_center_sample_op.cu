#include "hip/hip_runtime.h"
//   Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_HIP
#include <hiprand.h>
#include <hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
typedef hiprandState hiprandState;
namespace cub = hipcub;
#else
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
#endif

#include <iterator>
#include <random>
#include "paddle/fluid/operators/class_center_sample_op.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif

namespace paddle {
namespace operators {
#define CUDA_KERNEL_LOOP(i, n)                            \
  for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x, \
               step = blockDim.x * gridDim.x;             \
       i < (n); i += step)

using Tensor = framework::Tensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

inline int32_t NumBlocks(const int32_t n) {
  return std::min((n + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T>
__global__ void RandomSampleClassCenter(const int64_t n, int64_t seed,
                                        int64_t increment,
                                        const int64_t max_val, T* buffer) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState;
  size_t local_seed =
      (static_cast<size_t>(seed) + 0x9E3779B9U +
       (static_cast<size_t>(id) << 6U) + (static_cast<size_t>(id) >> 2U));
#ifdef PADDLE_WITH_HIP
  hiprand_init(local_seed, id, increment, &localState);
  CUDA_KERNEL_LOOP(i, n) {
    buffer[i] = static_cast<T>(hiprand(&localState) % max_val);
  }
#else
  hiprand_init(local_seed, id, increment, &localState);
  CUDA_KERNEL_LOOP(i, n) {
    buffer[i] = static_cast<T>(hiprand(&localState) % max_val);
  }
#endif
}

template <typename T>
__global__ void Range(const int64_t n, T* out) {
  CUDA_KERNEL_LOOP(i, n) { out[i] = static_cast<T>(i); }
}

template <typename T>
__global__ void MarkPositiveClassCenter(const int64_t n, const int64_t rank,
                                        const T* class_interval_ptr,
                                        const int num_classes, const T* labels,
                                        T* out) {
  CUDA_KERNEL_LOOP(i, n) {
    T label = labels[i] - class_interval_ptr[rank];
    if (label >= 0 && label < num_classes) {
      out[label] = label - num_classes;
    }
  }
}

template <typename T>
__device__ void FindIntervalIndex(const T* class_interval_ptr,
                                  const int64_t nranks, const T value,
                                  int64_t* find_index) {
  int64_t start = 0;
  int64_t end = nranks;
  int64_t mid = ((end - start) >> 1) + start + 1;
  while (start < end) {
    if (class_interval_ptr[mid] == value) break;
    if (class_interval_ptr[mid] > value)
      end = mid - 1;
    else
      start = mid;
    mid = ((end - start) >> 1) + start + 1;
  }
  *find_index = min(mid, end);
}

template <typename T>
__global__ void GetClassCenterBound(const int64_t n, const int64_t nranks,
                                    const T* class_interval_ptr,
                                    const T* key_ptr, const T* value_ptr,
                                    T* bound_index, T* bound_value) {
  CUDA_KERNEL_LOOP(i, n) {
    if (i != 0) {
      int64_t cur_index, pre_index;
      FindIntervalIndex(class_interval_ptr, nranks, key_ptr[i], &cur_index);
      FindIntervalIndex(class_interval_ptr, nranks, key_ptr[i - 1], &pre_index);
      if (cur_index > pre_index) {
        assert(cur_index < nranks);
#pragma unroll
        for (int32_t j = pre_index + 1; j <= cur_index; ++j) {
          bound_index[j] = static_cast<T>(i);
          bound_value[j] = value_ptr[i];
        }
      }
    }
  }
  CUDA_KERNEL_LOOP(i, nranks + 1) {
    int64_t first_index, last_index;
    FindIntervalIndex(class_interval_ptr, nranks, key_ptr[0], &first_index);
    FindIntervalIndex(class_interval_ptr, nranks, key_ptr[n - 1], &last_index);
    if (i <= first_index) {
      bound_index[i] = 0;
      bound_value[i] = value_ptr[0];
    } else if (i > last_index) {
      bound_index[i] = n;
      bound_value[i] = value_ptr[n - 1] + 1;
    }
  }
}

template <typename T>
__global__ void GetRemappedLabel(const int64_t n, const int64_t nranks,
                                 const T* sampled_class_interval_ptr,
                                 const T* bound_index, const T* bound_value,
                                 const T* label_map_key, T* label_map_value,
                                 T* mapped_label) {
  CUDA_KERNEL_LOOP(i, n) {
#pragma unroll
    for (int64_t j = 0; j < nranks; j++) {
      if (i >= bound_index[j] && i < bound_index[j + 1]) {
        label_map_value[i] =
            label_map_value[i] - bound_value[j] + sampled_class_interval_ptr[j];
      }
    }
    mapped_label[label_map_key[i]] = label_map_value[i];
  }
}

// aligned vector generates vectorized load/store on CUDA
template <typename T, int Size>
struct alignas(sizeof(T) * Size) AlignedVector {
  T val[Size];
};

template <typename T>
inline int VectorizedSize(const T* pointer) {
  uint64_t address = reinterpret_cast<uint64_t>(pointer);
  constexpr int vec4 = std::alignment_of<AlignedVector<T, 4>>::value;  // NOLINT
  if (address % vec4 == 0) {
    return 4;
  }
  return 1;
}

#undef CUDA_KERNEL_LOOP

template <typename T>
class NotEqualToPreviousAdjacentIterator {
 public:
  using self_type = NotEqualToPreviousAdjacentIterator;
  using value_type = T;
  using difference_type = std::ptrdiff_t;
  using pointer = T*;
  using reference = T;
  using iterator_category = std::input_iterator_tag;

 public:
  __host__ __device__ __forceinline__
  NotEqualToPreviousAdjacentIterator(const T* arr, int64_t offset)
      : arr_(arr), offset_(offset) {}

  __host__ __device__ __forceinline__ reference operator*() const {
    return offset_ == 0 ? 0 : (arr_[offset_] == arr_[offset_ - 1] ? 0 : 1);
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator+(Distance n) const {
    self_type ret(arr_, offset_ + n);
    return ret;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ reference operator[](Distance n) const {
    return *(*this + n);
  }

 private:
  const T* arr_;
  int64_t offset_;
};

template <typename T>
struct ActualNumSampledFunctor {
  __host__ __device__ __forceinline__ T operator()(const T& a,
                                                   const T& b) const {
    return max(num_samples, (b - a));
  }
  T num_samples;
  explicit ActualNumSampledFunctor(const T num) : num_samples(num) {}
};

template <typename T>
class MemoryBuffer {
 public:
  MemoryBuffer(const int num_buffer_ele, const int num_temp_ele,
               const int nranks, const platform::Place& place) {
    offset1 = 0;
    offset2 = offset1 + num_buffer_ele;
    offset3 = offset2 + num_buffer_ele;
    offset4 = offset3 + num_buffer_ele;
    offset5 = offset4 + num_buffer_ele;
    offset6 = offset5 + (nranks + 1);
    offset7 = offset6 + (nranks + 1);
    offset8 = offset7 + (nranks + 1);
    offset9 = offset8 + num_temp_ele;

    buffer_ptr = buffer.mutable_data<T>(
        {4 * num_buffer_ele + 3 * (nranks + 1) + num_temp_ele}, place);
  }

  T* cub_sort_keys_ptr() { return buffer_ptr + offset1; }
  T* cub_sort_keys_out_ptr() { return buffer_ptr + offset2; }
  T* cub_sort_values_ptr() { return buffer_ptr + offset3; }
  T* cub_sort_values_out_ptr() { return buffer_ptr + offset4; }
  T* bound_index_ptr() { return buffer_ptr + offset5; }
  T* bound_value_ptr() { return buffer_ptr + offset6; }
  T* class_interval_ptr() { return buffer_ptr + offset7; }
  void* cub_temp_storage_ptr() {
    return reinterpret_cast<void*>(buffer_ptr + offset8);
  }

 private:
  Tensor buffer;
  T* buffer_ptr;
  int offset1;
  int offset2;
  int offset3;
  int offset4;
  int offset5;
  int offset6;
  int offset7;
  int offset8;
  int offset9;
};

template <typename DeviceContext, typename T>
class ClassCenterSampleCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* label = ctx.Input<Tensor>("Label");
    auto* remapped_label = ctx.Output<Tensor>("RemappedLabel");
    auto* sampled_local_class_center =
        ctx.Output<Tensor>("SampledLocalClassCenter");
    int num_classes = ctx.Attr<int>("num_classes");
    int num_samples = ctx.Attr<int>("num_samples");

    int rid = ctx.Attr<int>("ring_id");
    int nranks = ctx.Attr<int>("nranks");
    int rank = ctx.Attr<int>("rank");

    int seed = ctx.Attr<int>("seed");
    bool fix_seed = ctx.Attr<bool>("fix_seed");
    PADDLE_ENFORCE_GT(num_classes, 0,
                      platform::errors::InvalidArgument(
                          "The value 'num_classes' for Op(class_center_sample) "
                          "must be greater than 0, "
                          "but the value given is %d.",
                          num_classes));

    PADDLE_ENFORCE_GT(num_samples, 0,
                      platform::errors::InvalidArgument(
                          "The value 'num_samples' for Op(class_center_sample) "
                          "must be greater than 0, "
                          "but the value given is %d.",
                          num_samples));

    PADDLE_ENFORCE_LE(num_samples, num_classes,
                      platform::errors::InvalidArgument(
                          "The value 'num_samples' for Op(class_center_sample) "
                          "must be less than or equal to %d, "
                          "but the value given is %d.",
                          num_classes, num_samples));

    auto& dev_ctx = ctx.template device_context<DeviceContext>();
    auto place = dev_ctx.GetPlace();

    int batch_size = label->numel();
    // Algorithm:
    // We first randomly generate a value in [0, num_classes) on each position
    // in a array(shape[num_classes]). Then, we mark the element as negative
    // value in the array according input label. Now, we can sort the array
    // by ascending to ensure that the positive class center always in the
    // front of the sorted array. So, we can get the sampled class center
    // index by sorted keys. Finally, we can get the rempped label by remap
    // the input label according sampled class center.

    // step 1: Calculate num classes per device using nccl all reduce
    std::vector<T> shard_dim_vec(nranks + 1, 0);
    shard_dim_vec[rank + 1] = num_classes;
    Tensor num_classes_per_device;
    framework::TensorFromVector(shard_dim_vec, ctx.cuda_device_context(),
                                &num_classes_per_device);
    T* num_classes_per_device_ptr = num_classes_per_device.data<T>();

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
    if (nranks > 1) {
      const auto& comm =
          platform::NCCLCommContext::Instance().Get(rid, ctx.GetPlace());
      // use global calculate stream
      const auto calcu_stream =
          static_cast<platform::CUDADeviceContext*>(
              platform::DeviceContextPool::Instance().Get(ctx.GetPlace()))
              ->stream();
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
          num_classes_per_device_ptr, num_classes_per_device_ptr,
          num_classes_per_device.numel(),
          platform::ToNCCLDataType(
              framework::TransToProtoVarType(num_classes_per_device.dtype())),
          ncclSum, comm->comm(), calcu_stream));
    }
#endif

    // step 2: Determine temporary device storage requirements
    int num_buffer_ele = std::max(batch_size, num_classes);
    size_t cub_sort_temp_store_size = 0;
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceRadixSort::SortPairs<T, T>(
        nullptr, cub_sort_temp_store_size, nullptr, nullptr, nullptr, nullptr,
        num_buffer_ele, 0, sizeof(T) * 8, ctx.cuda_device_context().stream())));

    size_t cub_sum_temp_store_size = 0;
    NotEqualToPreviousAdjacentIterator<T> unique_counting_iter_temp(nullptr, 0);
    PADDLE_ENFORCE_GPU_SUCCESS(
        (hipcub::DeviceScan::InclusiveSum<NotEqualToPreviousAdjacentIterator<T>,
                                       T*>(
            nullptr, cub_sum_temp_store_size, unique_counting_iter_temp,
            nullptr, batch_size, ctx.cuda_device_context().stream())));

    size_t cub_scan_temp_store_size = 0;
    ActualNumSampledFunctor<T> actual_num_sampled_op_temp(num_samples);
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceScan::InclusiveScan(
        nullptr, cub_scan_temp_store_size, num_classes_per_device_ptr,
        num_classes_per_device_ptr, actual_num_sampled_op_temp, nranks + 1,
        ctx.cuda_device_context().stream())));

    size_t cub_temp_storage_bytes =
        std::max(std::max(cub_sort_temp_store_size, cub_scan_temp_store_size),
                 cub_sum_temp_store_size);
    int num_temp_ele = cub_temp_storage_bytes / sizeof(T) + 1;

    // step 3: Alloc buffer memory so that we can reuse allocated memory
    MemoryBuffer<T> memory_buffer =
        MemoryBuffer<T>(num_buffer_ele, num_temp_ele, nranks, ctx.GetPlace());

    T* cub_sort_keys_ptr = memory_buffer.cub_sort_keys_ptr();
    T* cub_sort_keys_out_ptr = memory_buffer.cub_sort_keys_out_ptr();
    T* cub_sort_values_ptr = memory_buffer.cub_sort_values_ptr();
    T* cub_sort_values_out_ptr = memory_buffer.cub_sort_values_out_ptr();
    T* bound_index_ptr = memory_buffer.bound_index_ptr();
    T* bound_value_ptr = memory_buffer.bound_value_ptr();
    T* class_interval_ptr = memory_buffer.class_interval_ptr();
    void* cub_temp_storage_ptr = memory_buffer.cub_temp_storage_ptr();

    // step 4: Calculate class interval among nranks
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceScan::InclusiveSum(
        cub_temp_storage_ptr, cub_temp_storage_bytes,
        num_classes_per_device_ptr, class_interval_ptr, nranks + 1,
        ctx.cuda_device_context().stream())));

    // step 5: random sample negative class center
    uint64_t seed_data;
    uint64_t increment;
    int vec_size = VectorizedSize<T>(cub_sort_keys_ptr);
    auto offset = ((num_classes - 1) /
                       (NumBlocks(num_classes) * kNumCUDAThreads * vec_size) +
                   1) *
                  vec_size;
    int device_id = ctx.GetPlace().GetDeviceId();
    auto gen_cuda = framework::GetDefaultCUDAGenerator(device_id);
    if (gen_cuda->GetIsInitPy() && (!fix_seed)) {
      auto seed_offset = gen_cuda->IncrementOffset(offset);
      seed_data = seed_offset.first;
      increment = seed_offset.second;
    } else {
      std::random_device rnd;
      seed_data = fix_seed ? seed + rank : rnd();
      increment = offset;
    }
    RandomSampleClassCenter<T><<<NumBlocks(num_classes), kNumCUDAThreads, 0,
                                 ctx.cuda_device_context().stream()>>>(
        num_classes, seed_data, increment, num_classes, cub_sort_keys_ptr);

    // step 6: mark positive class center as negative value
    // fill the sort values to index 0, 1, ..., batch_size-1
    MarkPositiveClassCenter<<<NumBlocks(batch_size), kNumCUDAThreads, 0,
                              ctx.cuda_device_context().stream()>>>(
        batch_size, rank, class_interval_ptr, num_classes, label->data<T>(),
        cub_sort_keys_ptr);
    Range<T><<<NumBlocks(num_buffer_ele), kNumCUDAThreads, 0,
               ctx.cuda_device_context().stream()>>>(num_buffer_ele,
                                                     cub_sort_values_ptr);

    // step 7: sort class center by ascending, so that positive class center
    // always be sampled.
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceRadixSort::SortPairs<T, T>(
        cub_temp_storage_ptr, cub_temp_storage_bytes, cub_sort_keys_ptr,
        cub_sort_keys_out_ptr, cub_sort_values_ptr, cub_sort_values_out_ptr,
        num_classes, 0, sizeof(T) * 8, ctx.cuda_device_context().stream())));

    // step 8: sort input label ascending
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceRadixSort::SortPairs<T, T>(
        cub_temp_storage_ptr, cub_temp_storage_bytes, label->data<T>(),
        cub_sort_keys_out_ptr, cub_sort_values_ptr, cub_sort_keys_ptr,
        batch_size, 0, sizeof(T) * 8, ctx.cuda_device_context().stream())));

    // step 9: Calculate new index using InclusiveSum on ascending sorted input
    // label
    NotEqualToPreviousAdjacentIterator<T> unique_counting_iter(
        cub_sort_keys_out_ptr, 0);
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceScan::InclusiveSum<
                                NotEqualToPreviousAdjacentIterator<T>, T*>(
        cub_temp_storage_ptr, cub_temp_storage_bytes, unique_counting_iter,
        cub_sort_values_ptr, batch_size, ctx.cuda_device_context().stream())));

    // step 10: Calculate new class center bound among ranks
    GetClassCenterBound<T><<<NumBlocks(batch_size), kNumCUDAThreads, 0,
                             ctx.cuda_device_context().stream()>>>(
        batch_size, nranks, class_interval_ptr, cub_sort_keys_out_ptr,
        cub_sort_values_ptr, bound_index_ptr, bound_value_ptr);

    // step 11: Calculate actual number of sampled class per device.
    // Since maybe num_positive_class_center > num_samples,
    // we need to ensure all positive class center per device are sampled.
    ActualNumSampledFunctor<T> actual_num_sampled_op(num_samples);
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceScan::InclusiveScan(
        cub_temp_storage_ptr, cub_temp_storage_bytes, bound_value_ptr,
        num_classes_per_device_ptr, actual_num_sampled_op, nranks + 1,
        ctx.cuda_device_context().stream())));

    // step 12: Calculate actual sampled class interval among nranks
    PADDLE_ENFORCE_GPU_SUCCESS((hipcub::DeviceScan::InclusiveSum(
        cub_temp_storage_ptr, cub_temp_storage_bytes,
        num_classes_per_device_ptr, class_interval_ptr, nranks + 1,
        ctx.cuda_device_context().stream())));

    // step 13: Get remapped label for output
    GetRemappedLabel<T><<<NumBlocks(batch_size), kNumCUDAThreads, 0,
                          ctx.cuda_device_context().stream()>>>(
        batch_size, nranks, class_interval_ptr, bound_index_ptr,
        bound_value_ptr, cub_sort_keys_ptr, cub_sort_values_ptr,
        remapped_label->mutable_data<T>(ctx.GetPlace()));

    // step 14: Get sampled class center for output
    framework::TensorCopySync(num_classes_per_device, platform::CPUPlace(),
                              &num_classes_per_device);
    T actual_num_samples = num_classes_per_device.data<T>()[rank + 1];
    T* sampled_local_class_center_ptr =
        sampled_local_class_center->mutable_data<T>({actual_num_samples},
                                                    ctx.GetPlace());
    memory::Copy(place, sampled_local_class_center_ptr, place,
                 cub_sort_values_out_ptr, actual_num_samples * sizeof(T),
                 nullptr);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    class_center_sample,
    ops::ClassCenterSampleCUDAKernel<paddle::platform::CUDADeviceContext,
                                     int64_t>,
    ops::ClassCenterSampleCUDAKernel<paddle::platform::CUDADeviceContext, int>);
