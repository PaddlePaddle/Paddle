#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/random.h>
#include <thrust/transform.h>
#include <limits>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"

namespace paddle {
namespace operators {

template <typename T>
struct TruncatedNormal {
  T mean, std;
  T a_normal_cdf;
  T b_normal_cdf;
  unsigned int seed;
  T numeric_min;

  __host__ __device__ TruncatedNormal(T mean, T std, T numeric_min, int seed)
      : mean(mean), std(std), seed(seed), numeric_min(numeric_min) {
    a_normal_cdf = (1.0 + erff(-2.0 / sqrtf(2.0))) / 2.0;
    b_normal_cdf = (1.0 + erff(2.0 / sqrtf(2.0))) / 2.0;
  }

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed);
    thrust::uniform_real_distribution<T> dist(numeric_min, 1);
    rng.discard(n);
    T value = dist(rng);
    auto p = a_normal_cdf + (b_normal_cdf - a_normal_cdf) * value;
    return std::sqrt(2.0) * erfinvf(2 * p - 1) * std + mean;
  }
};

template <typename T>
class GPUTruncatedGaussianRandomKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* tensor = context.Output<framework::Tensor>("Out");
    T* data = tensor->mutable_data<T>(context.GetPlace());

    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
    }
    T mean = static_cast<T>(context.Attr<float>("mean"));
    T std = static_cast<T>(context.Attr<float>("std"));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    int64_t size = tensor->numel();
    thrust::transform(
        index_sequence_begin, index_sequence_begin + size,
        thrust::device_ptr<T>(data),
        TruncatedNormal<T>(mean, std, std::numeric_limits<T>::min(), seed));
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    truncated_gaussian_random,
    paddle::operators::GPUTruncatedGaussianRandomKernel<float>);
