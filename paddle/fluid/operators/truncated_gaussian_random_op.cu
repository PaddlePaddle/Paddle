#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <limits>
#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/truncated_gaussian_random_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct GPUTruncatedNormal {
  T mean, std;
  T a_normal_cdf;
  T b_normal_cdf;
  unsigned int seed;
  T numeric_min;

  __host__ __device__ GPUTruncatedNormal(T mean, T std, T numeric_min, int seed)
      : mean(mean), std(std), seed(seed), numeric_min(numeric_min) {
    a_normal_cdf = (1.0 + erff(-2.0 / sqrtf(2.0))) / 2.0;
    b_normal_cdf = (1.0 + erff(2.0 / sqrtf(2.0))) / 2.0;
  }

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed);
    thrust::uniform_real_distribution<T> dist(numeric_min, 1);
    rng.discard(n);
    T value = dist(rng);
    auto p = a_normal_cdf + (b_normal_cdf - a_normal_cdf) * value;
    return std::sqrt(2.0) * erfinvf(2 * p - 1) * std + mean;
  }
};

template <typename T>
struct TruncatedNormalOffset {
  T mean, std;
  T a_normal_cdf;
  T b_normal_cdf;
  unsigned int seed;
  T numeric_min;
  int offset_;

  __host__ __device__ TruncatedNormalOffset(T mean, T std, T numeric_min,
                                            int seed, int offset)
      : mean(mean),
        std(std),
        seed(seed),
        numeric_min(numeric_min),
        offset_(offset) {
    a_normal_cdf = (1.0 + erff(-2.0 / sqrtf(2.0))) / 2.0;
    b_normal_cdf = (1.0 + erff(2.0 / sqrtf(2.0))) / 2.0;
  }

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed);
    thrust::uniform_real_distribution<T> dist(numeric_min, 1);
    rng.discard(n + offset_);
    T value = dist(rng);
    auto p = a_normal_cdf + (b_normal_cdf - a_normal_cdf) * value;
    return std::sqrt(2.0) * erfinvf(2 * p - 1) * std + mean;
  }
};

template <typename T>
class GPUTruncatedGaussianRandomKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* tensor = context.Output<framework::Tensor>("Out");
    T* data = tensor->mutable_data<T>(context.GetPlace());

    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    bool seed_flag = false;
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
      seed_flag = true;
    }
    T mean = static_cast<T>(context.Attr<float>("mean"));
    T std = static_cast<T>(context.Attr<float>("std"));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    int64_t size = tensor->numel();

    int device_id =
        BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace()).GetDeviceId();
    auto gen_cuda = framework::GetDefaultCUDAGenerator(device_id);

    if (gen_cuda->GetIsInitPy() && seed_flag) {
      auto seed_offset = gen_cuda->IncrementOffset(1);
      int gen_offset = size * seed_offset.second;
      thrust::transform(
          index_sequence_begin, index_sequence_begin + size,
          thrust::device_ptr<T>(data),
          TruncatedNormalOffset<T>(mean, std, std::numeric_limits<T>::min(),
                                   seed_offset.first, gen_offset));
    } else {
      thrust::transform(index_sequence_begin, index_sequence_begin + size,
                        thrust::device_ptr<T>(data),
                        GPUTruncatedNormal<T>(
                            mean, std, std::numeric_limits<T>::min(), seed));
    }
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    truncated_gaussian_random,
    paddle::operators::GPUTruncatedGaussianRandomKernel<float>);
