#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/kernel_primitives/functor_primitives.h"
#include "paddle/fluid/operators/mean_op.h"
#include "paddle/fluid/operators/reduce_ops/reduce_op.cu.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void MeanRunKernel(const T* in_data, T* out_data, int N) {
  using MT = typename details::MPTypeTrait<T>::Type;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  auto data = static_cast<MT>(in_data[0]);
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    out_data[idx] = static_cast<T>(data / (static_cast<MT>(N)));
  }
}

template <typename DeviceContext, typename T>
class MeanCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<Tensor>("X");
    auto* output = context.Output<Tensor>("Out");

    const T* in_data = input->data<T>();
    T* out_data = output->mutable_data<T>(context.GetPlace());
    auto numel = input->numel();
    auto rank = input->dims().size();
    auto place = context.GetPlace();
    auto stream = context.cuda_device_context().stream();

    if (rank == 0) {  // scalar
      auto gpu_place = place;
      memory::Copy(gpu_place, out_data, gpu_place, in_data, numel * sizeof(T),
                   stream);
      return;
    }

    using Div = kernel_primitives::DivideFunctor<T, T>;
    std::vector<int> reduce_dims;
    reduce_dims.reserve(rank);
    for (decltype(rank) i = 0; i < rank; ++i) {
      reduce_dims.push_back(i);
    }
    TensorReduceImpl<T, T, kernel_primitives::AddFunctor, Div>(
        context.cuda_device_context(), *input, output, Div(numel), reduce_dims,
        stream);
  }
};

template <typename DeviceContext, typename T>
class MeanCUDAGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto OG = context.Input<Tensor>(framework::GradVarName("Out"));
    PADDLE_ENFORCE_EQ(OG->numel(), 1,
                      platform::errors::InvalidArgument(
                          "Mean Gradient Input Tensor len should be 1. But "
                          "received Out@Grad's elements num is %d.",
                          OG->numel()));
    auto IG = context.Output<Tensor>(framework::GradVarName("X"));
    IG->mutable_data<T>(context.GetPlace());

    auto in_data = OG->data<T>();
    auto size_prob = IG->numel();
    auto out_data = IG->data<T>();
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = context.cuda_device_context().stream();
    MeanRunKernel<T><<<grid, threads, 0, stream>>>(in_data, out_data,
                                                   size_prob);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    mean, ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    mean_grad,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
