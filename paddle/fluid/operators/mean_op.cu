#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/operators/mean_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n)
      : n_inv(static_cast<T>(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};

template <typename T>
__global__ void MeanRunKernel(const T* in_data, T* out_data, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  T data = in_data[0];
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    out_data[idx] = data / (static_cast<T>(N));
  }
}

template <typename DeviceContext, typename T>
class MeanCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<Tensor>("X");
    auto* output = context.Output<Tensor>("Out");

    output->mutable_data<T>(context.GetPlace());
    auto size_prob = input->numel();
    const T* in_data = input->data<T>();
    T* out_data = output->mutable_data<T>(context.GetPlace());
    auto stream = context.cuda_device_context().stream();

    DivideFunctor<T> transformer(size_prob);
    hipcub::TransformInputIterator<T, DivideFunctor<T>, const T*> trans_x(
        in_data, transformer);
    size_t temp_storage_bytes = 0;

    auto err = hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes, trans_x,
                                      out_data, size_prob, stream);
    PADDLE_ENFORCE_CUDA_SUCCESS(err);
    framework::Tensor tmp;
    auto* temp_storage = tmp.mutable_data<uint8_t>(
        framework::make_ddim({static_cast<int64_t>(temp_storage_bytes)}),
        context.GetPlace());
    err = hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, trans_x,
                                 out_data, size_prob, stream);
    PADDLE_ENFORCE_CUDA_SUCCESS(err);
  }
};

template <typename DeviceContext, typename T>
class MeanCUDAGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto OG = context.Input<Tensor>(framework::GradVarName("Out"));
    PADDLE_ENFORCE_EQ(OG->numel(), 1,
                      platform::errors::InvalidArgument(
                          "Mean Gradient Input Tensor len should be 1. But "
                          "received Out@Grad's elements num is %d.",
                          OG->numel()));
    auto IG = context.Output<Tensor>(framework::GradVarName("X"));
    IG->mutable_data<T>(context.GetPlace());

    auto in_data = OG->data<T>();
    auto size_prob = IG->numel();
    auto out_data = IG->data<T>();
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = context.cuda_device_context().stream();
    MeanRunKernel<T><<<grid, threads, 0, stream>>>(in_data, out_data,
                                                   size_prob);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    mean, ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    mean_grad,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
