#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/mean_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void MeanRunKernel(const T* in_data, T* out_data, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  T data = in_data[0];
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    out_data[idx] = data / (static_cast<T>(N));
  }
}

template <typename DeviceContext, typename T>
class MeanCUDAGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto OG = context.Input<Tensor>(framework::GradVarName("Out"));
    PADDLE_ENFORCE_EQ(OG->numel(), 1,
                      platform::errors::InvalidArgument(
                          "Mean Gradient Input Tensor len should be 1. But "
                          "received Out@Grad's elements num is %d.",
                          OG->numel()));
    auto IG = context.Output<Tensor>(framework::GradVarName("X"));
    IG->mutable_data<T>(context.GetPlace());

    auto in_data = OG->data<T>();
    auto size_prob = IG->numel();
    auto out_data = IG->data<T>();
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = context.cuda_device_context().stream();
    MeanRunKernel<T><<<grid, threads, 0, stream>>>(in_data, out_data,
                                                   size_prob);
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    mean, ops::MeanKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanKernel<paddle::platform::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    mean_grad,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::MeanCUDAGradKernel<paddle::platform::CUDADeviceContext,
                            plat::float16>);
