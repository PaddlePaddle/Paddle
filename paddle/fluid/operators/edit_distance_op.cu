#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/edit_distance_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

template <typename T>
__global__ void FillFirstRow(T* dist, const int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N + 1) {
    dist[idx] = idx;
  }
}

template <typename T>
__global__ void FillFirstColumn(T* dist, const int M, const int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < M + 1) {
    dist[idx * (N + 1)] = idx;
  }
}

template <typename T>
__global__ void Levenshtein(T* dist, const int64_t* x1, const int64_t* x2,
                            const int M, const int N, const int start) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = N;
  int index = start + idx * offset;
  int row = index / (N + 1);
  int col = index % (N + 1);
  if (row > 0 && col > 0 && row < M + 1 && col < N + 1) {
    int cost = x1[row - 1] == x2[col - 1] ? 0 : 1;
    int dels = dist[(row - 1) * (N + 1) + col] + 1;
    int ins = dist[row * (N + 1) + col - 1] + 1;
    int subs = dist[(row - 1) * (N + 1) + (col - 1)] + cost;
    dist[index] = min(dels, min(ins, subs));
  }
}

template <typename T>
__global__ void SetOutput(T* out, const T* dist, const int M, const int N,
                          bool normalized) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx == 0) {
    out[0] = normalized ? dist[M * (N + 1) + N] / N : dist[M * (N + 1) + N];
  }
}

template <typename Place, typename T>
class EditDistanceGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto* out_t = ctx.Output<framework::Tensor>("Out");

    auto* x1_t = ctx.Input<framework::LoDTensor>("Hyps");
    auto* x2_t = ctx.Input<framework::LoDTensor>("Refs");
    auto* sequence_num = ctx.Output<framework::Tensor>("SequenceNum");
    sequence_num->mutable_data<int64_t>(ctx.GetPlace());
    auto batch_size = x1_t->dims()[0];

    auto normalized = ctx.Attr<bool>("normalized");
    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();

    framework::Vector<size_t> hyp_lod(batch_size + 1);
    framework::Vector<size_t> ref_lod(batch_size + 1);

    bool use_length = ctx.HasInput("HypsLength");

    if (use_length) {
      // build lod when using padding
      auto* hyp_length = ctx.Input<framework::Tensor>("HypsLength");
      auto* ref_length = ctx.Input<framework::Tensor>("RefsLength");

      framework::Tensor hyp_length_cpu;
      framework::Tensor ref_length_cpu;
      framework::TensorCopy(*hyp_length, platform::CPUPlace(), &hyp_length_cpu);
      framework::TensorCopy(*ref_length, platform::CPUPlace(), &ref_length_cpu);

      for (auto i = 0; i < batch_size; i++) {
        hyp_lod[i + 1] = hyp_lod[i] + hyp_length_cpu.data<int64_t>()[i];
        ref_lod[i + 1] = ref_lod[i] + ref_length_cpu.data<int64_t>()[i];
      }

    } else {
      hyp_lod = x1_t->lod()[0];
      ref_lod = x2_t->lod()[0];
    }

    if (normalized) {
      for (size_t i = 1; i < ref_lod.size(); ++i) {
        PADDLE_ENFORCE(ref_lod[i] > ref_lod[i - 1],
                       "Reference string %d is empty.", i);
      }
    }

    const size_t num_strs = hyp_lod.size() - 1;
    math::SetConstant<platform::CUDADeviceContext, int64_t> set_constant;
    set_constant(ctx.template device_context<platform::CUDADeviceContext>(),
                 sequence_num, static_cast<int64_t>(num_strs));

    out_t->Resize({static_cast<int64_t>(num_strs), 1});
    out_t->mutable_data<T>(ctx.GetPlace());
    auto out = out_t->data<T>();

    T distance = 0.0;
    for (size_t num = 0; num < num_strs; num++) {
      auto m = static_cast<int64_t>(hyp_lod[num + 1] - hyp_lod[num]);
      auto n = static_cast<int64_t>(ref_lod[num + 1] - ref_lod[num]);
      if (m == 0 || n == 0) {
        distance = std::max(m, n);
        if (normalized) {
          distance = distance / n;
        }
        memory::Copy(BOOST_GET_CONST(Place, ctx.GetPlace()), out + num,
                     platform::CPUPlace(), &distance, sizeof(T), stream);
      } else {
        framework::Tensor dist_t;
        dist_t.Resize({m + 1, n + 1});
        dist_t.mutable_data<T>(ctx.GetPlace());
        auto dist = dist_t.data<T>();
        auto hyp_offset = use_length ? num * x1_t->dims()[1] : hyp_lod[num];
        auto ref_offset = use_length ? num * x2_t->dims()[1] : ref_lod[num];
        auto x1 = x1_t->data<int64_t>() + hyp_offset;
        auto x2 = x2_t->data<int64_t>() + ref_offset;

        FillFirstColumn<T><<<1 + m / PADDLE_CUDA_NUM_THREADS,
                             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, m, n);

        FillFirstRow<T><<<1 + n / PADDLE_CUDA_NUM_THREADS,
                          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, n);

        // Compute the elements of distance matrix in the anti-diagonal diretion
        for (int64_t slice = 2; slice < m + n + 1; ++slice) {
          int z_m = slice < m + 1 ? 0 : slice - m;
          int z_n = slice < n + 1 ? 0 : slice - n;
          int size = slice - (z_m + z_n) + 1;  // number of elments in the same
                                               // anti-diagonal line to update
          // the start index at which computes from
          int start = slice < n + 1 ? slice : (z_n + 1) * (n + 1) - 1;
          Levenshtein<T><<<1 + (size - 1) / PADDLE_CUDA_NUM_THREADS,
                           PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, x1, x2,
                                                                 m, n, start);
        }
        SetOutput<T><<<1, 1, 0, stream>>>(out + num, dist, m, n, normalized);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    edit_distance,
    ops::EditDistanceGPUKernel<paddle::platform::CUDAPlace, float>);
