#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/diagonal_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

template <typename T, int INPUT_DIM_SIZE, int OUTPUT_DIM_SIZE>
__global__ void Diagonal(const T* input_data, T* output_data,
                         const int64_t offset_, int64_t axis1_, int64_t axis2_,
                         int64_t* input_stride, int64_t* output_stride,
                         int64_t numel) {
  CUDA_KERNEL_LOOP(idx, numel) {
    int64_t idx_dim[INPUT_DIM_SIZE] = {0};
    int64_t temp = 0;
    for (size_t i = 0; i < INPUT_DIM_SIZE - 1; i++) {
      idx_dim[i] = (idx - temp) / input_stride[i];
      temp = temp + idx_dim[i] * input_stride[i];
    }
    idx_dim[INPUT_DIM_SIZE - 1] = idx - temp;

    int64_t axis1_dim = idx_dim[axis1_];
    int64_t axis2_dim = idx_dim[axis2_];

    int64_t out_dim[OUTPUT_DIM_SIZE] = {0};
    int temp_pos = 0;
    for (int i = 0; i < INPUT_DIM_SIZE; i++) {
      if (i != axis1_ && i != axis2_) {
        out_dim[temp_pos] = idx_dim[i];
        temp_pos++;
      }
    }
    bool flag = false;
    if (offset_ == 0 && axis1_dim == axis2_dim) {
      out_dim[temp_pos] = axis1_dim;
      flag = true;
    } else if (offset_ > 0 && (axis1_dim + offset_) == axis2_dim) {
      out_dim[temp_pos] = axis1_dim;
      flag = true;
    } else if (offset_ < 0 && (axis1_dim + offset_) == axis2_dim) {
      out_dim[temp_pos] = axis2_dim;
      flag = true;
    }
    if (flag) {
      int64_t idx_output = 0;
      for (size_t i = 0; i < OUTPUT_DIM_SIZE - 1; i++) {
        idx_output = idx_output + out_dim[i] * output_stride[i];
      }
      idx_output = idx_output + out_dim[OUTPUT_DIM_SIZE - 1];
      output_data[idx_output] = input_data[idx];
    }
  }
}

template <typename T>
class DiagonalCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<framework::Tensor>("Input");
    const auto* input_data = input->data<T>();
    auto input_dim = input->dims().Get();
    auto input_dim_size = input->dims().size();

    int input_stride_size = input_dim_size - 1;
    int64_t* host_input_stride;
    host_input_stride =
        reinterpret_cast<int64_t*>(malloc(input_stride_size * sizeof(int64_t)));
    for (size_t i_input = 0; i_input < input_stride_size; i_input++) {
      int64_t temp_stride = 1;
      for (size_t j = i_input + 1; j < input_dim_size; j++) {
        temp_stride = temp_stride * input_dim[j];
      }
      host_input_stride[i_input] = temp_stride;
    }
    int64_t* input_stride;
    hipMalloc(reinterpret_cast<void**>(&input_stride),
               input_stride_size * sizeof(int64_t));
    hipMemcpy(reinterpret_cast<void*>(input_stride),
               reinterpret_cast<void*>(host_input_stride),
               input_stride_size * sizeof(int64_t), hipMemcpyHostToDevice);

    auto* output = context.Output<framework::Tensor>("Out");
    auto* output_data = output->mutable_data<T>(context.GetPlace());
    auto output_dim = output->dims().Get();
    auto output_dim_size = output->dims().size();

    int output_stride_size = output_dim_size - 1;
    int64_t* host_output_stride;
    host_output_stride = reinterpret_cast<int64_t*>(
        malloc(output_stride_size * sizeof(int64_t)));
    for (size_t i_output = 0; i_output < output_stride_size; i_output++) {
      int64_t temp_stride = 1;
      for (size_t j = i_output + 1; j < output_dim_size; j++) {
        temp_stride = temp_stride * output_dim[j];
      }
      host_output_stride[i_output] = temp_stride;
    }
    int64_t* output_stride;
    hipMalloc(reinterpret_cast<void**>(&output_stride),
               output_stride_size * sizeof(int64_t));
    hipMemcpy(reinterpret_cast<void*>(output_stride),
               reinterpret_cast<void*>(host_output_stride),
               output_stride_size * sizeof(int64_t), hipMemcpyHostToDevice);

    const int64_t offset_ = context.Attr<int>("offset");
    const int64_t axis1 = context.Attr<int>("axis1");
    int64_t axis1_ = axis1 < 0 ? input_dim_size + axis1 : axis1;
    const int64_t axis2 = context.Attr<int>("axis2");
    int64_t axis2_ = axis2 < 0 ? input_dim_size + axis2 : axis2;
    int64_t numel = input->numel();

    int threads = PADDLE_CUDA_NUM_THREADS;
    int blocks = (numel + threads - 1) / threads;

    switch (input_dim_size) {
      case 2:
        Diagonal<T, 2, 1><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 3:
        Diagonal<T, 3, 2><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 4:
        Diagonal<T, 4, 3><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 5:
        Diagonal<T, 5, 4><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 6:
        Diagonal<T, 6, 5><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 7:
        Diagonal<T, 7, 6><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 8:
        Diagonal<T, 8, 7><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      case 9:
        Diagonal<T, 9, 8><<<blocks, threads>>>(input_data, output_data, offset_,
                                               axis1_, axis2_, input_stride,
                                               output_stride, numel);
        break;
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "The rank of input should be less than 10, but received %d.",
            input_dim_size));
    }
    hipFree(input_stride);
    hipFree(output_stride);
    free(host_input_stride);
    free(host_output_stride);
  }
};

template <typename T, int DX_DIM_SIZE, int DOUT_DIM_SIZE>
__global__ void GradDiagonal(const T* dout_data, T* dx_data,
                             const int64_t offset_, int64_t axis1_,
                             int64_t axis2_, int64_t* dout_stride,
                             int64_t* dx_stride, int64_t numel) {
  CUDA_KERNEL_LOOP(idx, numel) {
    int64_t idx_dim[DX_DIM_SIZE] = {0};
    int64_t temp = 0;
    for (size_t i = 0; i < DX_DIM_SIZE - 1; i++) {
      idx_dim[i] = (idx - temp) / dx_stride[i];
      temp = temp + idx_dim[i] * dx_stride[i];
    }
    idx_dim[DX_DIM_SIZE - 1] = idx - temp;

    int64_t axis1_dim = idx_dim[axis1_];
    int64_t axis2_dim = idx_dim[axis2_];

    int64_t dout_dim[DOUT_DIM_SIZE] = {0};
    int temp_pos = 0;
    for (int i = 0; i < DX_DIM_SIZE; i++) {
      if (i != axis1_ && i != axis2_) {
        dout_dim[temp_pos] = idx_dim[i];
        temp_pos++;
      }
    }
    bool flag = false;
    if (offset_ == 0 && axis1_dim == axis2_dim) {
      dout_dim[temp_pos] = axis1_dim;
      flag = true;
    } else if (offset_ > 0 && (axis1_dim + offset_) == axis2_dim) {
      dout_dim[temp_pos] = axis1_dim;
      flag = true;
    } else if (offset_ < 0 && (axis1_dim + offset_) == axis2_dim) {
      dout_dim[temp_pos] = axis2_dim;
      flag = true;
    }
    if (flag) {
      int64_t idx_output = 0;
      for (size_t i = 0; i < DOUT_DIM_SIZE - 1; i++) {
        idx_output = idx_output + dout_dim[i] * dout_stride[i];
      }
      idx_output = idx_output + dout_dim[DOUT_DIM_SIZE - 1];
      dx_data[idx] = dout_data[idx_output];
    } else {
      dx_data[idx] = static_cast<T>(0);
    }
  }
}

template <typename T>
class DiagonalGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    const auto* dout =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    const auto* dout_data = dout->data<T>();
    auto dout_dim = dout->dims().Get();
    auto dout_dim_size = dout->dims().size();

    int dout_stride_size = dout_dim_size - 1;
    int64_t* host_dout_stride;
    host_dout_stride =
        reinterpret_cast<int64_t*>(malloc(dout_stride_size * sizeof(int64_t)));
    for (size_t i_dout = 0; i_dout < dout_stride_size; i_dout++) {
      int64_t temp_stride = 1;
      for (size_t j = i_dout + 1; j < dout_dim_size; j++) {
        temp_stride = temp_stride * dout_dim[j];
      }
      host_dout_stride[i_dout] = temp_stride;
    }
    int64_t* dout_stride;
    hipMalloc(reinterpret_cast<void**>(&dout_stride),
               dout_stride_size * sizeof(int64_t));
    hipMemcpy(reinterpret_cast<void*>(dout_stride),
               reinterpret_cast<void*>(host_dout_stride),
               dout_stride_size * sizeof(int64_t), hipMemcpyHostToDevice);

    auto* dx =
        context.Output<framework::Tensor>(framework::GradVarName("Input"));
    auto* dx_data = dx->mutable_data<T>(context.GetPlace());
    auto dx_dim = dx->dims().Get();
    auto dx_dim_size = dx->dims().size();

    int dx_stride_size = dx_dim_size - 1;
    int64_t* host_dx_stride;
    host_dx_stride =
        reinterpret_cast<int64_t*>(malloc(dx_stride_size * sizeof(int64_t)));
    for (size_t i_dx = 0; i_dx < dx_stride_size; i_dx++) {
      int64_t temp_stride = 1;
      for (size_t j = i_dx + 1; j < dx_dim_size; j++) {
        temp_stride = temp_stride * dx_dim[j];
      }
      host_dx_stride[i_dx] = temp_stride;
    }
    int64_t* dx_stride;
    hipMalloc(reinterpret_cast<void**>(&dx_stride),
               dx_stride_size * sizeof(int64_t));
    hipMemcpy(reinterpret_cast<void*>(dx_stride),
               reinterpret_cast<void*>(host_dx_stride),
               dx_stride_size * sizeof(int64_t), hipMemcpyHostToDevice);

    const int64_t offset_ = context.Attr<int>("offset");
    const int64_t axis1 = context.Attr<int>("axis1");
    int64_t axis1_ = axis1 < 0 ? dx_dim_size + axis1 : axis1;
    const int64_t axis2 = context.Attr<int>("axis2");
    int64_t axis2_ = axis2 < 0 ? dx_dim_size + axis2 : axis2;

    int64_t numel = dx->numel();

    int threads = PADDLE_CUDA_NUM_THREADS;
    int blocks = (numel + threads - 1) / threads;

    switch (dx_dim_size) {
      case 2:
        GradDiagonal<T, 2, 1><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 3:
        GradDiagonal<T, 3, 2><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 4:
        GradDiagonal<T, 4, 3><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 5:
        GradDiagonal<T, 5, 4><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 6:
        GradDiagonal<T, 6, 5><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 7:
        GradDiagonal<T, 7, 6><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 8:
        GradDiagonal<T, 8, 7><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      case 9:
        GradDiagonal<T, 9, 8><<<blocks, threads>>>(dout_data, dx_data, offset_,
                                                   axis1_, axis2_, dout_stride,
                                                   dx_stride, numel);
        break;
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "The rank of output(input@Grad) should be less than 10, but "
            "received %d.",
            dx_dim_size));
    }
    hipFree(dout_stride);
    hipFree(dx_stride);
    free(host_dout_stride);
    free(host_dx_stride);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(diagonal, ops::DiagonalCUDAKernel<int>,
                        ops::DiagonalCUDAKernel<int64_t>,
                        ops::DiagonalCUDAKernel<float>,
                        ops::DiagonalCUDAKernel<double>);

REGISTER_OP_CUDA_KERNEL(diagonal_grad, ops::DiagonalGradCUDAKernel<int>,
                        ops::DiagonalGradCUDAKernel<int64_t>,
                        ops::DiagonalGradCUDAKernel<float>,
                        ops::DiagonalGradCUDAKernel<double>);
