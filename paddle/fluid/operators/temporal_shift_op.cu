#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/fluid/operators/temporal_shift_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace operators {

using framework::Tensor;

template <typename T>
__global__ void KeTemporalShiftFw(const T* input, T* output, const int ntchw,
                                  const int tchw, const int chw, const int hw,
                                  const int w, const int t, const int c,
                                  const float shift_ratio) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int src_it = 0;
  for (; tid < ntchw; tid += stride) {
    int in = tid / tchw;
    int it = (tid % tchw) / chw;
    int ic = (tid % chw) / hw;
    int ih = (tid % hw) / w;
    int iw = tid % w;

    const int c1 = static_cast<int>(c * shift_ratio);
    const int c2 = static_cast<int>(c * 2 * shift_ratio);

    if (ic < c1) {
      src_it = it - 1;
    } else if (ic < c2) {
      src_it = it + 1;
    } else {
      src_it = it;
    }

    if (src_it < 0 || src_it >= t) {
      output[tid] = 0;
    } else {
      int src_idx = GetEntryIndex(in, src_it, ic, ih, iw, tchw, chw, hw, w);
      output[tid] = input[src_idx];
    }
  }
}

template <typename T>
__global__ void KeTemporalShiftBw(const T* output_grad, T* input_grad,
                                  const int ntchw, const int tchw,
                                  const int chw, const int hw, const int w,
                                  const int t, const int c,
                                  const float shift_ratio) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int src_it = 0;
  for (; tid < ntchw; tid += stride) {
    int in = tid / tchw;
    int it = (tid % tchw) / chw;
    int ic = (tid % chw) / hw;
    int ih = (tid % hw) / w;
    int iw = tid % w;

    const int c1 = static_cast<int>(c * shift_ratio);
    const int c2 = static_cast<int>(c * 2 * shift_ratio);

    if (ic < c1) {
      src_it = it - 1;
    } else if (ic < c2) {
      src_it = it + 1;
    } else {
      src_it = it;
    }

    if (src_it >= 0 && src_it < t) {
      int src_idx = GetEntryIndex(in, src_it, ic, ih, iw, tchw, chw, hw, w);
      input_grad[src_idx] = output_grad[tid];
    }
  }
}

template <typename T>
class TemporalShiftOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx.GetPlace()), true,
                      platform::errors::InvalidArgument(
                          "This kernel only runs on GPU device."));
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");
    int t = ctx.Attr<int>("seg_num");
    float shift_ratio = ctx.Attr<float>("shift_ratio");

    const int nt = input->dims()[0];
    const int c = input->dims()[1];
    const int h = input->dims()[2];
    const int w = input->dims()[3];

    const int hw = h * w;
    const int chw = c * hw;
    const int tchw = t * chw;
    const int ntchw = nt * chw;

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>({nt, c, h, w}, ctx.GetPlace());

    int pixelNum = nt * chw;
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

    KeTemporalShiftFw<T><<<config.block_per_grid, config.thread_per_block, 0,
                           ctx.cuda_device_context().stream()>>>(
        input_data, output_data, ntchw, tchw, chw, hw, w, t, c, shift_ratio);
  }
};

template <typename T>
class TemporalShiftGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    int t = ctx.Attr<int>("seg_num");
    float shift_ratio = ctx.Attr<float>("shift_ratio");

    const int nt = output_grad->dims()[0];
    const int c = output_grad->dims()[1];
    const int h = output_grad->dims()[2];
    const int w = output_grad->dims()[3];

    const int hw = h * w;
    const int chw = c * hw;
    const int tchw = t * chw;
    const int ntchw = nt * chw;

    const T* output_grad_data = output_grad->data<T>();
    T* input_grad_data =
        input_grad->mutable_data<T>({nt, c, h, w}, ctx.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, T>()(
        ctx.template device_context<platform::CUDADeviceContext>(), input_grad,
        static_cast<T>(0));

    int pixelNum = nt * chw;
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

    KeTemporalShiftBw<T><<<config.block_per_grid, config.thread_per_block, 0,
                           ctx.cuda_device_context().stream()>>>(
        output_grad_data, input_grad_data, ntchw, tchw, chw, hw, w, t, c,
        shift_ratio);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    temporal_shift, ops::TemporalShiftOpCUDAKernel<float>,
    ops::TemporalShiftOpCUDAKernel<double>,
    ops::TemporalShiftOpCUDAKernel<paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    temporal_shift_grad, ops::TemporalShiftGradOpCUDAKernel<float>,
    ops::TemporalShiftGradOpCUDAKernel<double>,
    ops::TemporalShiftGradOpCUDAKernel<paddle::platform::float16>);
