#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include <paddle/fluid/memory/allocation/allocator.h>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/detection/bbox_util.h"
#include "paddle/fluid/operators/detection/distribute_fpn_proposals_op.h"
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 64;
static constexpr int kNumMaxinumNumBlocks = 4096;

int const BBoxSize = 4;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <class T>
__global__ void GPUDistFpnProposalsHelper(
    const int nthreads, const T* rois, const int lod_size,
    const int refer_level, const int refer_scale, const int max_level,
    const int min_level, int* roi_batch_id_data, int* sub_lod_list,
    int* target_lvls, bool pixel_offset = true) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const T* offset_roi = rois + i * BBoxSize;
    int roi_batch_ind = roi_batch_id_data[i];
    // get the target level of current rois
    T roi_area = RoIArea(offset_roi, pixel_offset);
    T roi_scale = sqrt(roi_area);
    int tgt_lvl = floor(
        log2(roi_scale / static_cast<T>(refer_scale) + (T)1e-8) + refer_level);
    tgt_lvl = min(max_level, max(tgt_lvl, min_level));
    target_lvls[i] = tgt_lvl;
    // compute number of rois in the same batch and same target level
    platform::CudaAtomicAdd(
        sub_lod_list + (tgt_lvl - min_level) * lod_size + roi_batch_ind, 1);
  }
}

template <typename DeviceContext, typename T>
class GPUDistributeFpnProposalsOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* fpn_rois = ctx.Input<paddle::framework::LoDTensor>("FpnRois");

    auto multi_fpn_rois = ctx.MultiOutput<LoDTensor>("MultiFpnRois");
    auto* restore_index = ctx.Output<Tensor>("RestoreIndex");

    const int min_level = ctx.Attr<int>("min_level");
    const int max_level = ctx.Attr<int>("max_level");
    const int refer_level = ctx.Attr<int>("refer_level");
    const int refer_scale = ctx.Attr<int>("refer_scale");
    const bool pixel_offset = ctx.Attr<bool>("pixel_offset");
    int num_level = max_level - min_level + 1;

    // check that the fpn_rois is not empty
    if (!ctx.HasInput("RoisNum")) {
      PADDLE_ENFORCE_EQ(
          fpn_rois->lod().size(), 1UL,
          platform::errors::InvalidArgument("DistributeFpnProposalsOp needs LoD"
                                            "with one level"));
    }

    std::vector<size_t> fpn_rois_lod;
    if (ctx.HasInput("RoisNum")) {
      auto* rois_num = ctx.Input<Tensor>("RoisNum");
      fpn_rois_lod = GetLodFromRoisNum(rois_num);
    } else {
      fpn_rois_lod = fpn_rois->lod().back();
    }
    int lod_size = fpn_rois_lod.size() - 1;
    int roi_num = fpn_rois_lod[lod_size];

    auto& dev_ctx = ctx.template device_context<DeviceContext>();

    // get batch id by lod in CPU
    Tensor roi_batch_id_list;
    roi_batch_id_list.Resize({roi_num});
    int* roi_batch_id_data =
        roi_batch_id_list.mutable_data<int>(platform::CPUPlace());
    for (int n = 0; n < lod_size; ++n) {
      for (size_t i = fpn_rois_lod[n]; i < fpn_rois_lod[n + 1]; ++i) {
        roi_batch_id_data[i] = n;
      }
    }
    // copy batch id list to GPU
    Tensor roi_batch_id_list_gpu;
    framework::TensorCopySync(roi_batch_id_list, dev_ctx.GetPlace(),
                              &roi_batch_id_list_gpu);

    Tensor sub_lod_list;
    sub_lod_list.Resize({num_level, lod_size});
    int* sub_lod_list_data = sub_lod_list.mutable_data<int>(dev_ctx.GetPlace());
    math::SetConstant<platform::CUDADeviceContext, int> set_zero;
    set_zero(dev_ctx, &sub_lod_list, static_cast<int>(0));

    Tensor target_lvls;
    target_lvls.Resize({roi_num});
    int* target_lvls_data = target_lvls.mutable_data<int>(dev_ctx.GetPlace());

    int dist_blocks = NumBlocks(roi_num);
    int threads = kNumCUDAThreads;
    // get target levels and sub_lod list
    GPUDistFpnProposalsHelper<T><<<dist_blocks, threads>>>(
        roi_num, fpn_rois->data<T>(), lod_size, refer_level, refer_scale,
        max_level, min_level, roi_batch_id_list_gpu.data<int>(),
        sub_lod_list_data, target_lvls_data, pixel_offset);
    dev_ctx.Wait();
    auto place = BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace());

    Tensor index_in_t;
    int* idx_in = index_in_t.mutable_data<int>({roi_num}, dev_ctx.GetPlace());
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx, roi_num);
    for_range(RangeInitFunctor{0, 1, idx_in});

    Tensor keys_out_t;
    int* keys_out = keys_out_t.mutable_data<int>({roi_num}, dev_ctx.GetPlace());
    Tensor index_out_t;
    int* idx_out = index_out_t.mutable_data<int>({roi_num}, dev_ctx.GetPlace());

    // Determine temporary device storage requirements
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairs<int, int>(nullptr, temp_storage_bytes,
                                              target_lvls_data, keys_out,
                                              idx_in, idx_out, roi_num);
    // Allocate temporary storage
    auto d_temp_storage = memory::Alloc(place, temp_storage_bytes);

    // Run sorting operation
    // sort target level to get corresponding index
    hipcub::DeviceRadixSort::SortPairs<int, int>(
        d_temp_storage->ptr(), temp_storage_bytes, target_lvls_data, keys_out,
        idx_in, idx_out, roi_num);

    int* restore_idx_data =
        restore_index->mutable_data<int>({roi_num, 1}, dev_ctx.GetPlace());
    // sort current index to get restore index
    hipcub::DeviceRadixSort::SortPairs<int, int>(
        d_temp_storage->ptr(), temp_storage_bytes, idx_out, keys_out, idx_in,
        restore_idx_data, roi_num);

    int start = 0;
    auto multi_rois_num = ctx.MultiOutput<Tensor>("MultiLevelRoIsNum");

    for (int i = 0; i < num_level; ++i) {
      Tensor sub_lod = sub_lod_list.Slice(i, i + 1);
      int* sub_lod_data = sub_lod.data<int>();
      // transfer length-based lod to offset-based lod
      std::vector<size_t> offset(1, 0);
      std::vector<int> sub_lod_cpu(lod_size);
      memory::Copy(platform::CPUPlace(), sub_lod_cpu.data(), place,
                   sub_lod_data, sizeof(int) * lod_size, dev_ctx.stream());
      dev_ctx.Wait();
      for (int j = 0; j < lod_size; ++j) {
        offset.emplace_back(offset.back() + sub_lod_cpu[j]);
      }

      int sub_rois_num = offset.back();

      int end = start + sub_rois_num;
      if (end > start) {
        Tensor sub_idx = index_out_t.Slice(start, end);
        start = end;
        multi_fpn_rois[i]->mutable_data<T>({sub_rois_num, kBoxDim},
                                           dev_ctx.GetPlace());
        GPUGather<T>(dev_ctx, *fpn_rois, sub_idx, multi_fpn_rois[i]);
      } else {
        multi_fpn_rois[i]->mutable_data<T>({sub_rois_num, kBoxDim},
                                           dev_ctx.GetPlace());
      }
      if (multi_rois_num.size() > 0) {
        Tensor* rois_num_t = multi_rois_num[i];
        TensorCopySync(sub_lod, dev_ctx.GetPlace(), rois_num_t);
        rois_num_t->Resize({lod_size});
      }
      framework::LoD lod;
      lod.emplace_back(offset);
      multi_fpn_rois[i]->set_lod(lod);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    distribute_fpn_proposals,
    ops::GPUDistributeFpnProposalsOpKernel<paddle::platform::CUDADeviceContext,
                                           float>,
    ops::GPUDistributeFpnProposalsOpKernel<paddle::platform::CUDADeviceContext,
                                           double>);
