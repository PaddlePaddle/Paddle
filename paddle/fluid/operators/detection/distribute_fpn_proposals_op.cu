#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/memory/allocation/allocator.h>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/detection/distribute_fpn_proposals_op.h"
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

int const BBoxSize = 4;

struct RangeInitFunctor {
  int start_;
  int delta_;
  int* out_;
  __device__ void operator()(size_t i) { out_[i] = start_ + i * delta_; }
};

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

static inline void TransLoD(const int* length_lod, const int lod_size,
                            int* offset_lod) {
  int offset = 0;
  for (int i = 0; i < lod_size; ++i) {
    offset_lod[i] = offset;
    offset += length_lod[i];
  }
}

template <typename T>
static __device__ inline T RoIArea(const T* box, bool normalized) {
  if (box[2] < box[0] || box[3] < box[1]) {
    // If coordinate values are is invalid
    // (e.g. xmax < xmin or ymax < ymin), return 0.
    return static_cast<T>(0.);
  } else {
    const T w = box[2] - box[0];
    const T h = box[3] - box[1];
    if (normalized) {
      return w * h;
    } else {
      // If coordinate values are not within range [0, 1].
      return (w + 1) * (h + 1);
    }
  }
}

template <class T>
static __global__ void GPUDistFpnProposalsHelper(
    const int nthreads, const T* rois, const int lod_size,
    const int refer_level, const int refer_scale, const int max_level,
    const int min_level, int* roi_batch_id_data, int* sub_lod_list,
    int* target_lvls) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    const T* offset_roi = rois + i * BBoxSize;
    int roi_batch_ind = roi_batch_id_data[i];
    // get the target level of current rois
    T roi_area = RoIArea(offset_roi, false);
    T roi_scale = sqrt(roi_area);
    int tgt_lvl = floor(log2(roi_scale / refer_scale) + refer_level);
    tgt_lvl = min(max_level, max(tgt_lvl, min_level));
    target_lvls[i] = tgt_lvl;
    // compute number of rois in the same batch and same target level
    platform::CudaAtomicAdd(sub_lod_list + tgt_lvl * lod_size + roi_batch_ind,
                            1);
  }
}

template <typename DeviceContext, typename T>
class GPUDistributeFpnProposalsOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* fpn_rois = ctx.Input<paddle::framework::LoDTensor>("FpnRois");

    auto multi_fpn_rois = ctx.MultiOutput<LoDTensor>("MultiFpnRois");
    auto* restore_index = ctx.Output<Tensor>("RestoreIndex");

    const int min_level = ctx.Attr<int>("min_level");
    const int max_level = ctx.Attr<int>("max_level");
    const int refer_level = ctx.Attr<int>("refer_level");
    const int refer_scale = ctx.Attr<int>("refer_scale");
    int num_level = max_level - min_level + 1;

    // check that the fpn_rois is not empty
    PADDLE_ENFORCE_EQ(fpn_rois->lod().size(), 1UL,
                      "DistributeFpnProposalsOp need 1 level of LoD");

    auto fpn_rois_lod = fpn_rois->lod().back();
    int lod_size = fpn_rois_lod.size() - 1;
    int roi_num = fpn_rois_lod[lod_size];

    auto& dev_ctx = ctx.template device_context<DeviceContext>();

    // get batch id by lod in CPU
    Tensor roi_batch_id_list;
    roi_batch_id_list.Resize({roi_num});
    int* roi_batch_id_data =
        roi_batch_id_list.mutable_data<int>(platform::CPUPlace());
    for (int n = 0; n < lod_size; ++n) {
      for (size_t i = fpn_rois_lod[n]; i < fpn_rois_lod[n + 1]; ++i) {
        roi_batch_id_data[i] = n;
      }
    }
    // copy batch id list to GPU
    Tensor roi_batch_id_list_gpu;
    framework::TensorCopySync(roi_batch_id_list, dev_ctx.GetPlace(),
                              &roi_batch_id_list_gpu);

    Tensor sub_lod_list;
    sub_lod_list.Resize({num_level, lod_size});
    int* sub_lod_list_data = sub_lod_list.mutable_data<int>(dev_ctx.GetPlace());
    Tensor target_lvls;
    target_lvls.Resize({roi_num});
    int* target_lvls_data = target_lvls.mutable_data<int>(dev_ctx.GetPlace());

    int blocks = NumBlocks(roi_num);
    int threads = kNumCUDAThreads;

    // get target levels and sub_lod list
    GPUDistFpnProposalsHelper<T><<<blocks, threads>>>(
        roi_num, fpn_rois->data<T>(), lod_size, refer_level, refer_scale,
        max_level, min_level, roi_batch_id_list_gpu.data<int>(),
        sub_lod_list_data, target_lvls_data);

    Tensor index_in_t;
    int* idx_in = index_in_t.mutable_data<int>({roi_num}, dev_ctx.GetPlace());
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx, roi_num);
    for_range(RangeInitFunctor{0, 1, idx_in});

    Tensor keys_out_t;
    int* keys_out = keys_out_t.mutable_data<int>({roi_num}, dev_ctx.GetPlace());
    Tensor index_out_t;
    int* idx_out = index_out_t.mutable_data<int>({roi_num}, dev_ctx.GetPlace());

    // Determine temporary device storage requirements
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairsDescending<int, int>(
        nullptr, temp_storage_bytes, target_lvls_data, keys_out, idx_in,
        idx_out, roi_num);
    // Allocate temporary storage
    auto place = boost::get<platform::CUDAPlace>(dev_ctx.GetPlace());
    auto d_temp_storage = memory::Alloc(place, temp_storage_bytes,
                                        memory::Allocator::kScratchpad);

    // Run sorting operation
    // sort target level to get corresponding index
    hipcub::DeviceRadixSort::SortPairsDescending<int, int>(
        d_temp_storage->ptr(), temp_storage_bytes, target_lvls_data, keys_out,
        idx_in, idx_out, roi_num);

    int* restore_idx_data =
        restore_index->mutable_data<int>({roi_num, 1}, dev_ctx.GetPlace());
    // sort current index to get restore index
    hipcub::DeviceRadixSort::SortPairsDescending<int, int>(
        d_temp_storage->ptr(), temp_storage_bytes, idx_out, keys_out, idx_in,
        restore_idx_data, roi_num);

    Tensor offset_lod;
    int* offset_lod_data =
        offset_lod.mutable_data<int>({lod_size + 1}, dev_ctx.GetPlace());
    for (int i = 0; i < num_level; ++i) {
      Tensor sub_lod = sub_lod_list.Slice(i, i + 1);
      int* sub_lod_data = sub_lod.data<int>();
      // transfer length-based lod to offset-based lod
      TransLoD(sub_lod_data, lod_size + 1, offset_lod_data);
      int sub_rois_num = offset_lod_data[lod_size];
      Tensor sub_idx = index_out_t.Slice(0, sub_rois_num);

      multi_fpn_rois[i]->mutable_data<T>({sub_rois_num, kBoxDim},
                                         dev_ctx.GetPlace());

      GPUGather<T>(dev_ctx, *fpn_rois, sub_idx, multi_fpn_rois[i]);
      framework::LoD lod;
      std::vector<size_t> offset;
      memory::Copy(platform::CPUPlace(), offset.data(), place, offset_lod_data,
                   sizeof(int) * (lod_size + 1), 0);
      lod.emplace_back(offset);
      multi_fpn_rois[i]->set_lod(lod);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    distribute_fpn_proposals,
    ops::GPUDistributeFpnProposalsOpKernel<paddle::platform::CUDADeviceContext,
                                           float>,
    ops::GPUDistributeFpnProposalsOpKernel<paddle::platform::CUDADeviceContext,
                                           double>);
