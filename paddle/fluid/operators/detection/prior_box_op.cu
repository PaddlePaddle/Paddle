#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/detection/prior_box_op.h"

namespace paddle {
namespace operators {

template <typename T>
__device__ inline T clip(T in) {
  return min(max(in, 0.), 1.);
}

template <typename T>
__global__ void GenPriorBox(T* out, const T* aspect_ratios, const int height,
                            const int width, const int im_height,
                            const int im_width, const int as_num,
                            const T offset, const T step_width,
                            const T step_height, const T* min_sizes,
                            const T* max_sizes, const int min_num, bool is_clip,
                            bool min_max_aspect_ratios_order) {
  int num_priors = max_sizes ? as_num * min_num + min_num : as_num * min_num;
  int box_num = height * width * num_priors;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < box_num;
       i += blockDim.x * gridDim.x) {
    int h = i / (num_priors * width);
    int w = (i / num_priors) % width;
    int p = i % num_priors;
    int m = max_sizes ? p / (as_num + 1) : p / as_num;
    T cx = (w + offset) * step_width;
    T cy = (h + offset) * step_height;
    T bw, bh;
    T min_size = min_sizes[m];
    if (max_sizes) {
      int s = p % (as_num + 1);
      if (!min_max_aspect_ratios_order) {
        if (s < as_num) {
          T ar = aspect_ratios[s];
          bw = min_size * sqrt(ar) / 2.;
          bh = min_size / sqrt(ar) / 2.;
        } else {
          T max_size = max_sizes[m];
          bw = sqrt(min_size * max_size) / 2.;
          bh = bw;
        }
      } else {
        if (s == 0) {
          bw = bh = min_size / 2.;
        } else if (s == 1) {
          T max_size = max_sizes[m];
          bw = sqrt(min_size * max_size) / 2.;
          bh = bw;
        } else {
          T ar = aspect_ratios[s - 1];
          bw = min_size * sqrt(ar) / 2.;
          bh = min_size / sqrt(ar) / 2.;
        }
      }
    } else {
      int s = p % as_num;
      T ar = aspect_ratios[s];
      bw = min_size * sqrt(ar) / 2.;
      bh = min_size / sqrt(ar) / 2.;
    }
    T xmin = (cx - bw) / im_width;
    T ymin = (cy - bh) / im_height;
    T xmax = (cx + bw) / im_width;
    T ymax = (cy + bh) / im_height;
    out[i * 4] = is_clip ? clip<T>(xmin) : xmin;
    out[i * 4 + 1] = is_clip ? clip<T>(ymin) : ymin;
    out[i * 4 + 2] = is_clip ? clip<T>(xmax) : xmax;
    out[i * 4 + 3] = is_clip ? clip<T>(ymax) : ymax;
  }
}

template <typename T>
__global__ void SetVariance(T* out, const T* var, const int vnum,
                            const int num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    out[i] = var[i % vnum];
  }
}

template <typename T>
class PriorBoxOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<paddle::framework::Tensor>("Input");
    auto* image = ctx.Input<paddle::framework::Tensor>("Image");
    auto* boxes = ctx.Output<paddle::framework::Tensor>("Boxes");
    auto* vars = ctx.Output<paddle::framework::Tensor>("Variances");

    auto min_sizes = ctx.Attr<std::vector<float>>("min_sizes");
    auto max_sizes = ctx.Attr<std::vector<float>>("max_sizes");
    auto input_aspect_ratio = ctx.Attr<std::vector<float>>("aspect_ratios");
    auto variances = ctx.Attr<std::vector<float>>("variances");
    auto flip = ctx.Attr<bool>("flip");
    auto clip = ctx.Attr<bool>("clip");
    auto min_max_aspect_ratios_order =
        ctx.Attr<bool>("min_max_aspect_ratios_order");

    std::vector<float> aspect_ratios;
    ExpandAspectRatios(input_aspect_ratio, flip, &aspect_ratios);

    T step_w = static_cast<T>(ctx.Attr<float>("step_w"));
    T step_h = static_cast<T>(ctx.Attr<float>("step_h"));
    T offset = static_cast<T>(ctx.Attr<float>("offset"));

    auto im_width = image->dims()[3];
    auto im_height = image->dims()[2];

    auto width = input->dims()[3];
    auto height = input->dims()[2];

    T step_width, step_height;
    if (step_w == 0 || step_h == 0) {
      step_width = static_cast<T>(im_width) / width;
      step_height = static_cast<T>(im_height) / height;
    } else {
      step_width = step_w;
      step_height = step_h;
    }

    int num_priors = aspect_ratios.size() * min_sizes.size();
    if (max_sizes.size() > 0) {
      num_priors += max_sizes.size();
    }
    int min_num = static_cast<int>(min_sizes.size());
    int box_num = width * height * num_priors;

    int block = 512;
    int grid = (box_num + block - 1) / block;

    auto stream =
        ctx.template device_context<platform::CUDADeviceContext>().stream();

    boxes->mutable_data<T>(ctx.GetPlace());
    vars->mutable_data<T>(ctx.GetPlace());

    framework::Tensor r;
    framework::TensorFromVector(aspect_ratios, ctx.device_context(), &r);

    framework::Tensor min;
    framework::TensorFromVector(min_sizes, ctx.device_context(), &min);

    T* max_data = nullptr;
    framework::Tensor max;
    if (max_sizes.size() > 0) {
      framework::TensorFromVector(max_sizes, ctx.device_context(), &max);
      max_data = max.data<T>();
    }

    GenPriorBox<T><<<grid, block, 0, stream>>>(
        boxes->data<T>(), r.data<T>(), height, width, im_height, im_width,
        aspect_ratios.size(), offset, step_width, step_height, min.data<T>(),
        max_data, min_num, clip, min_max_aspect_ratios_order);

    framework::Tensor v;
    framework::TensorFromVector(variances, ctx.device_context(), &v);
    grid = (box_num * 4 + block - 1) / block;
    SetVariance<T><<<grid, block, 0, stream>>>(vars->data<T>(), v.data<T>(),
                                               variances.size(), box_num * 4);
  }
};  // namespace operators

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(prior_box, ops::PriorBoxOpCUDAKernel<float>,
                        ops::PriorBoxOpCUDAKernel<double>);
