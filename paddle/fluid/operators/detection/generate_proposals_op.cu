#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/memory/allocation/allocator.h>
#include <stdio.h>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

namespace {

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))
#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

int const kThreadsPerBlock = sizeof(uint64_t) * 8;

static const double kBBoxClipDefault = std::log(1000.0 / 16.0);

struct RangeInitFunctor {
  int start_;
  int delta_;
  int *out_;
  __device__ void operator()(size_t i) { out_[i] = start_ + i * delta_; }
};

template <typename T>
static void SortDescending(const platform::CUDADeviceContext &ctx,
                           const Tensor &value, Tensor *value_out,
                           Tensor *index_out) {
  int num = static_cast<int>(value.numel());
  Tensor index_in_t;
  int *idx_in = index_in_t.mutable_data<int>({num}, ctx.GetPlace());
  platform::ForRange<platform::CUDADeviceContext> for_range(ctx, num);
  for_range(RangeInitFunctor{0, 1, idx_in});

  int *idx_out = index_out->mutable_data<int>({num}, ctx.GetPlace());

  const T *keys_in = value.data<T>();
  T *keys_out = value_out->mutable_data<T>({num}, ctx.GetPlace());

  // Determine temporary device storage requirements
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending<T, int>(
      nullptr, temp_storage_bytes, keys_in, keys_out, idx_in, idx_out, num);
  // Allocate temporary storage
  auto place = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());
  auto d_temp_storage = memory::Alloc(place, temp_storage_bytes);

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairsDescending<T, int>(
      d_temp_storage->ptr(), temp_storage_bytes, keys_in, keys_out, idx_in,
      idx_out, num);
}

template <typename T>
struct BoxDecodeAndClipFunctor {
  const T *anchor;
  const T *deltas;
  const T *var;
  const int *index;
  const T *im_info;

  T *proposals;

  BoxDecodeAndClipFunctor(const T *anchor, const T *deltas, const T *var,
                          const int *index, const T *im_info, T *proposals)
      : anchor(anchor),
        deltas(deltas),
        var(var),
        index(index),
        im_info(im_info),
        proposals(proposals) {}

  T bbox_clip_default{static_cast<T>(kBBoxClipDefault)};

  __device__ void operator()(size_t i) {
    int k = index[i] * 4;
    T axmin = anchor[k];
    T aymin = anchor[k + 1];
    T axmax = anchor[k + 2];
    T aymax = anchor[k + 3];

    T w = axmax - axmin + 1.0;
    T h = aymax - aymin + 1.0;
    T cx = axmin + 0.5 * w;
    T cy = aymin + 0.5 * h;

    T dxmin = deltas[k];
    T dymin = deltas[k + 1];
    T dxmax = deltas[k + 2];
    T dymax = deltas[k + 3];

    T d_cx, d_cy, d_w, d_h;
    if (var) {
      d_cx = cx + dxmin * w * var[k];
      d_cy = cy + dymin * h * var[k + 1];
      d_w = exp(Min(dxmax * var[k + 2], bbox_clip_default)) * w;
      d_h = exp(Min(dymax * var[k + 3], bbox_clip_default)) * h;
    } else {
      d_cx = cx + dxmin * w;
      d_cy = cy + dymin * h;
      d_w = exp(Min(dxmax, bbox_clip_default)) * w;
      d_h = exp(Min(dymax, bbox_clip_default)) * h;
    }

    T oxmin = d_cx - d_w * 0.5;
    T oymin = d_cy - d_h * 0.5;
    T oxmax = d_cx + d_w * 0.5 - 1.;
    T oymax = d_cy + d_h * 0.5 - 1.;

    proposals[i * 4] = Max(Min(oxmin, im_info[1] - 1.), 0.);
    proposals[i * 4 + 1] = Max(Min(oymin, im_info[0] - 1.), 0.);
    proposals[i * 4 + 2] = Max(Min(oxmax, im_info[1] - 1.), 0.);
    proposals[i * 4 + 3] = Max(Min(oymax, im_info[0] - 1.), 0.);
  }

  __device__ __forceinline__ T Min(T a, T b) const { return a > b ? b : a; }

  __device__ __forceinline__ T Max(T a, T b) const { return a > b ? a : b; }
};

template <typename T, int BlockSize>
static __global__ void FilterBBoxes(const T *bboxes, const T *im_info,
                                    const T min_size, const int num,
                                    int *keep_num, int *keep) {
  T im_h = im_info[0];
  T im_w = im_info[1];
  T im_scale = im_info[2];

  int cnt = 0;
  __shared__ int keep_index[BlockSize];

  CUDA_1D_KERNEL_LOOP(i, num) {
    keep_index[threadIdx.x] = -1;
    __syncthreads();

    int k = i * 4;
    T xmin = bboxes[k];
    T ymin = bboxes[k + 1];
    T xmax = bboxes[k + 2];
    T ymax = bboxes[k + 3];

    T w = xmax - xmin + 1.0;
    T h = ymax - ymin + 1.0;
    T cx = xmin + w / 2.;
    T cy = ymin + h / 2.;

    T w_s = (xmax - xmin) / im_scale + 1.;
    T h_s = (ymax - ymin) / im_scale + 1.;

    if (w_s >= min_size && h_s >= min_size && cx <= im_w && cy <= im_h) {
      keep_index[threadIdx.x] = i;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
      int size = (num - i) < BlockSize ? num - i : BlockSize;
      for (int j = 0; j < size; ++j) {
        if (keep_index[j] > -1) {
          keep[cnt++] = keep_index[j];
        }
      }
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    keep_num[0] = cnt;
  }
}

static __device__ inline float IoU(const float *a, const float *b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float inter_s = width * height;
  float s_a = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float s_b = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return inter_s / (s_a + s_b - inter_s);
}

static __global__ void NMSKernel(const int n_boxes,
                                 const float nms_overlap_thresh,
                                 const float *dev_boxes, uint64_t *dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  const int row_size =
      min(n_boxes - row_start * kThreadsPerBlock, kThreadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * kThreadsPerBlock, kThreadsPerBlock);

  __shared__ float block_boxes[kThreadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[(kThreadsPerBlock * col_start + threadIdx.x) * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[(kThreadsPerBlock * col_start + threadIdx.x) * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[(kThreadsPerBlock * col_start + threadIdx.x) * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[(kThreadsPerBlock * col_start + threadIdx.x) * 4 + 3];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = kThreadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 4;
    int i = 0;
    uint64_t t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (IoU(cur_box, block_boxes + i * 4) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, kThreadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

template <typename T>
static void NMS(const platform::CUDADeviceContext &ctx, const Tensor &proposals,
                const Tensor &sorted_indices, const T nms_threshold,
                Tensor *keep_out) {
  int boxes_num = proposals.dims()[0];
  PADDLE_ENFORCE_EQ(boxes_num, sorted_indices.dims()[0]);

  const int col_blocks = DIVUP(boxes_num, kThreadsPerBlock);
  dim3 blocks(DIVUP(boxes_num, kThreadsPerBlock),
              DIVUP(boxes_num, kThreadsPerBlock));
  dim3 threads(kThreadsPerBlock);

  const T *boxes = proposals.data<T>();
  auto place = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());
  framework::Vector<uint64_t> mask(boxes_num * col_blocks);
  NMSKernel<<<blocks, threads>>>(boxes_num, nms_threshold, boxes,
                                 mask.CUDAMutableData(BOOST_GET_CONST(
                                     platform::CUDAPlace, ctx.GetPlace())));

  std::vector<uint64_t> remv(col_blocks);
  memset(&remv[0], 0, sizeof(uint64_t) * col_blocks);

  std::vector<int> keep_vec;
  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / kThreadsPerBlock;
    int inblock = i % kThreadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      ++num_to_keep;
      keep_vec.push_back(i);
      uint64_t *p = &mask[0] + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  int *keep = keep_out->mutable_data<int>({num_to_keep}, ctx.GetPlace());
  memory::Copy(place, keep, platform::CPUPlace(), keep_vec.data(),
               sizeof(int) * num_to_keep, ctx.stream());
  ctx.Wait();
}

template <typename T>
static std::pair<Tensor, Tensor> ProposalForOneImage(
    const platform::CUDADeviceContext &ctx, const Tensor &im_info,
    const Tensor &anchors, const Tensor &variances,
    const Tensor &bbox_deltas,  // [M, 4]
    const Tensor &scores,       // [N, 1]
    int pre_nms_top_n, int post_nms_top_n, float nms_thresh, float min_size,
    float eta) {
  // 1. pre nms
  Tensor scores_sort, index_sort;
  SortDescending<T>(ctx, scores, &scores_sort, &index_sort);
  int num = scores.numel();
  int pre_nms_num = (pre_nms_top_n <= 0 || pre_nms_top_n > num) ? scores.numel()
                                                                : pre_nms_top_n;
  scores_sort.Resize({pre_nms_num, 1});
  index_sort.Resize({pre_nms_num, 1});

  // 2. box decode and clipping
  Tensor proposals;
  proposals.mutable_data<T>({pre_nms_num, 4}, ctx.GetPlace());

  {
    platform::ForRange<platform::CUDADeviceContext> for_range(ctx, pre_nms_num);
    for_range(BoxDecodeAndClipFunctor<T>{
        anchors.data<T>(), bbox_deltas.data<T>(), variances.data<T>(),
        index_sort.data<int>(), im_info.data<T>(), proposals.data<T>()});
  }

  // 3. filter
  Tensor keep_index, keep_num_t;
  keep_index.mutable_data<int>({pre_nms_num}, ctx.GetPlace());
  keep_num_t.mutable_data<int>({1}, ctx.GetPlace());
  min_size = std::max(min_size, 1.0f);
  auto stream = ctx.stream();
  FilterBBoxes<T, 512><<<1, 512, 0, stream>>>(
      proposals.data<T>(), im_info.data<T>(), min_size, pre_nms_num,
      keep_num_t.data<int>(), keep_index.data<int>());
  int keep_num;
  const auto gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());
  memory::Copy(platform::CPUPlace(), &keep_num, gpu_place,
               keep_num_t.data<int>(), sizeof(int), ctx.stream());
  ctx.Wait();
  keep_index.Resize({keep_num});

  Tensor scores_filter, proposals_filter;
  proposals_filter.mutable_data<T>({keep_num, 4}, ctx.GetPlace());
  scores_filter.mutable_data<T>({keep_num, 1}, ctx.GetPlace());
  GPUGather<T>(ctx, proposals, keep_index, &proposals_filter);
  GPUGather<T>(ctx, scores_sort, keep_index, &scores_filter);

  if (nms_thresh <= 0) {
    return std::make_pair(proposals_filter, scores_filter);
  }

  // 4. nms
  Tensor keep_nms;
  NMS<T>(ctx, proposals_filter, keep_index, nms_thresh, &keep_nms);
  if (post_nms_top_n > 0 && post_nms_top_n < keep_nms.numel()) {
    keep_nms.Resize({post_nms_top_n});
  }

  Tensor scores_nms, proposals_nms;
  proposals_nms.mutable_data<T>({keep_nms.numel(), 4}, ctx.GetPlace());
  scores_nms.mutable_data<T>({keep_nms.numel(), 1}, ctx.GetPlace());
  GPUGather<T>(ctx, proposals_filter, keep_nms, &proposals_nms);
  GPUGather<T>(ctx, scores_filter, keep_nms, &scores_nms);

  return std::make_pair(proposals_nms, scores_nms);
}
}  // namespace

template <typename DeviceContext, typename T>
class CUDAGenerateProposalsKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *scores = context.Input<Tensor>("Scores");
    auto *bbox_deltas = context.Input<Tensor>("BboxDeltas");
    auto *im_info = context.Input<Tensor>("ImInfo");
    auto anchors = GET_DATA_SAFELY(context.Input<Tensor>("Anchors"), "Input",
                                   "Anchors", "GenerateProposals");
    auto variances = GET_DATA_SAFELY(context.Input<Tensor>("Variances"),
                                     "Input", "Variances", "GenerateProposals");

    auto *rpn_rois = context.Output<LoDTensor>("RpnRois");
    auto *rpn_roi_probs = context.Output<LoDTensor>("RpnRoiProbs");

    int pre_nms_top_n = context.Attr<int>("pre_nms_topN");
    int post_nms_top_n = context.Attr<int>("post_nms_topN");
    float nms_thresh = context.Attr<float>("nms_thresh");
    float min_size = context.Attr<float>("min_size");
    float eta = context.Attr<float>("eta");
    PADDLE_ENFORCE_GE(eta, 1., "Not support adaptive NMS.");

    auto &dev_ctx = context.template device_context<DeviceContext>();

    auto scores_dim = scores->dims();
    int64_t num = scores_dim[0];
    int64_t c_score = scores_dim[1];
    int64_t h_score = scores_dim[2];
    int64_t w_score = scores_dim[3];

    auto bbox_dim = bbox_deltas->dims();
    int64_t c_bbox = bbox_dim[1];
    int64_t h_bbox = bbox_dim[2];
    int64_t w_bbox = bbox_dim[3];

    Tensor bbox_deltas_swap, scores_swap;
    bbox_deltas_swap.mutable_data<T>({num, h_bbox, w_bbox, c_bbox},
                                     dev_ctx.GetPlace());
    scores_swap.mutable_data<T>({num, h_score, w_score, c_score},
                                dev_ctx.GetPlace());

    math::Transpose<DeviceContext, T, 4> trans;
    std::vector<int> axis = {0, 2, 3, 1};
    trans(dev_ctx, *bbox_deltas, &bbox_deltas_swap, axis);
    trans(dev_ctx, *scores, &scores_swap, axis);

    anchors.Resize({anchors.numel() / 4, 4});
    variances.Resize({variances.numel() / 4, 4});

    rpn_rois->mutable_data<T>({bbox_deltas->numel() / 4, 4},
                              context.GetPlace());
    rpn_roi_probs->mutable_data<T>({scores->numel(), 1}, context.GetPlace());

    T *rpn_rois_data = rpn_rois->data<T>();
    T *rpn_roi_probs_data = rpn_roi_probs->data<T>();

    auto place = BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace());
    auto cpu_place = platform::CPUPlace();

    int64_t num_proposals = 0;
    std::vector<size_t> offset(1, 0);
    std::vector<int64_t> tmp_lod;

    for (int64_t i = 0; i < num; ++i) {
      Tensor im_info_slice = im_info->Slice(i, i + 1);
      Tensor bbox_deltas_slice = bbox_deltas_swap.Slice(i, i + 1);
      Tensor scores_slice = scores_swap.Slice(i, i + 1);

      bbox_deltas_slice.Resize({h_bbox * w_bbox * c_bbox / 4, 4});
      scores_slice.Resize({h_score * w_score * c_score, 1});

      std::pair<Tensor, Tensor> box_score_pair =
          ProposalForOneImage<T>(dev_ctx, im_info_slice, anchors, variances,
                                 bbox_deltas_slice, scores_slice, pre_nms_top_n,
                                 post_nms_top_n, nms_thresh, min_size, eta);

      Tensor &proposals = box_score_pair.first;
      Tensor &scores = box_score_pair.second;

      memory::Copy(place, rpn_rois_data + num_proposals * 4, place,
                   proposals.data<T>(), sizeof(T) * proposals.numel(),
                   dev_ctx.stream());
      memory::Copy(place, rpn_roi_probs_data + num_proposals, place,
                   scores.data<T>(), sizeof(T) * scores.numel(),
                   dev_ctx.stream());
      dev_ctx.Wait();
      num_proposals += proposals.dims()[0];
      offset.emplace_back(num_proposals);
      tmp_lod.push_back(num_proposals);
    }
    if (context.HasOutput("RpnRoisLod")) {
      auto *rpn_rois_lod = context.Output<Tensor>("RpnRoisLod");
      rpn_rois_lod->mutable_data<int64_t>({num}, context.GetPlace());
      int64_t *lod_data = rpn_rois_lod->data<int64_t>();
      memory::Copy(place, lod_data, cpu_place, &tmp_lod[0],
                   sizeof(int64_t) * num, dev_ctx.stream());
      rpn_rois_lod->Resize({num});
    }
    framework::LoD lod;
    lod.emplace_back(offset);
    rpn_rois->set_lod(lod);
    rpn_roi_probs->set_lod(lod);
    rpn_rois->Resize({num_proposals, 4});
    rpn_roi_probs->Resize({num_proposals, 1});
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(generate_proposals,
                        ops::CUDAGenerateProposalsKernel<
                            paddle::platform::CUDADeviceContext, float>);
