#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/detection/yolo_box_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
static __global__ void GenDensityPriorBox(
    const int height, const int width, const int im_height, const int im_width,
    const T offset, const T step_width, const T step_height,
    const int num_priors, const T* ratios_shift, bool is_clip, const T var_xmin,
    const T var_ymin, const T var_xmax, const T var_ymax, T* out, T* var) {
  int gidx = blockIdx.x * blockDim.x + threadIdx.x;
  int gidy = blockIdx.y * blockDim.y + threadIdx.y;
  int step_x = blockDim.x * gridDim.x;
  int step_y = blockDim.y * gridDim.y;
}

template <typename T>
class YoloBoxOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<Tensor>("Input");
    auto* boxes = ctx.Output<Tensor>("Boxes");
    auto* scores = ctx.Output<Tensor>("Scores");

    auto anchors = ctx.Attr<std::vector<int>>("anchors");
    int class_num = ctx.Attr<int>("class_num");
    float conf_thresh = ctx.Attr<float>("conf_thresh");
    int downsample_ratio = ctx.Attr<int>("downsample_ratio");

    const int n = input->dims()[0];
    const int h = input->dims()[2];
    const int w = input->dims()[3];
    const int box_num = boxes->dims()[1];
    const int an_num = anchors.size() / 2;
    int input_size = downsample_ratio * h;

    const int stride = h * w;
    const int an_stride = (class_num + 5) * stride;

    const T* input_data = input->data<T>();
    T* boxes_data = boxes->mutable_data<T>({n}, ctx.GetPlace());
    memset(loss_data, 0, boxes->numel() * sizeof(T));
    T* scores_data = scores->mutable_data<T>({n}, ctx.GetPlace());
    memset(scores_data, 0, scores->numel() * sizeof(T));
  }
};  // namespace operators

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(density_prior_box,
                        ops::DensityPriorBoxOpCUDAKernel<float>,
                        ops::DensityPriorBoxOpCUDAKernel<double>);
