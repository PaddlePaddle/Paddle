#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/detection/yolo_box_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void KeYoloBoxFw(const T* input, const int* imgsize, T* boxes,
                            T* scores, const float conf_thresh,
                            std::vector<int> anchors, const int h, const in w,
                            const int an_num, const int class_num,
                            const int box_num, const int input_size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < box_num; tid += stride) {
    int grid_num = h * w;
    int i = tid / box_num;
    int j = (tid % box_num) / grid_num;
    int k = (tid % grid_num) / w;
    int l = tid % w;

    int an_stride = an_num * grid_num;
    int img_height = imgsize[2 * i];
    int img_width = imgsize[2 * i + 1];

    int obj_idx =
        GetEntryIndex(i, j, k * w + l, an_num, an_stride, grid_num, 4);
    T conf = sigmoid<T>(input[obj_idx]);
    if (conf < conf_thresh) {
      continue;
    }

    int box_idx =
        GetEntryIndex(i, j, k * w + l, an_num, an_stride, grid_num, 0);
    Box<T> pred = GetYoloBox<T>(input, anchors, l, k, j, h, input_size, box_idx,
                                grid_num, img_height, img_width);
    box_idx = (i * box_num + j * grid_num + k * w + l) * 4;
    CalcDetectionBox<T>(boxes, pred, box_idx);

    int label_idx =
        GetEntryIndex(i, j, k * w + l, an_num, an_stride, grid_num, 5);
    int score_idx = (i * box_num + j * stride + k * w + l) * class_num;
    CalcLabelScore<T>(scores, input, label_idx, score_idx, class_num, conf,
                      grid_num);
  }
}

template <typename T>
class YoloBoxOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<Tensor>("Input");
    auto* img_size = ctx.Input<Tensor>("ImgSize");
    auto* boxes = ctx.Output<Tensor>("Boxes");
    auto* scores = ctx.Output<Tensor>("Scores");

    auto anchors = ctx.Attr<std::vector<int>>("anchors");
    int class_num = ctx.Attr<int>("class_num");
    float conf_thresh = ctx.Attr<float>("conf_thresh");
    int downsample_ratio = ctx.Attr<int>("downsample_ratio");

    const int n = input->dims()[0];
    const int h = input->dims()[2];
    const int w = input->dims()[3];
    const int box_num = boxes->dims()[1];
    const int an_num = anchors.size() / 2;
    int input_size = downsample_ratio * h;

    const T* input_data = input->data<T>();
    const int* imgsize_data = imgsize->data<int>();
    T* boxes_data = boxes->mutable_data<T>({n, box_num, 4}, ctx.GetPlace());
    memset(boxes_data, 0, boxes->numel() * sizeof(T));
    T* scores_data =
        scores->mutable_data<T>({n, box_num, class_num}, ctx.GetPlace());
    memset(scores_data, 0, scores->numel() * sizeof(T));

    int grid_dim = (n * box_num + 512 - 1) / 512;
    grid_dim = grid_dim > 8 ? 8 : grid_dim;
  }
};  // namespace operators

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(density_prior_box,
                        ops::DensityPriorBoxOpCUDAKernel<float>,
                        ops::DensityPriorBoxOpCUDAKernel<double>);
