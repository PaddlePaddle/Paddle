#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/detection/anchor_generator_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void GenAnchors(T* out, const T* aspect_ratios, const int ar_num,
                           const T* anchor_sizes, const int as_num,
                           const T* stride, const int sd_num, const int height,
                           const int width, const T offset) {
  int num_anchors = as_num * ar_num;
  int box_num = height * width * num_anchors;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < box_num;
       i += blockDim.x * gridDim.x) {
    int h_idx = i / (num_anchors * width);
    int w_idx = (i / num_anchors) % width;
    T stride_width = stride[0];
    T stride_height = stride[1];
    T x_ctr = (w_idx * stride_width) + offset * (stride_width - 1);
    T y_ctr = (h_idx * stride_height) + offset * (stride_height - 1);
    T area, area_ratios;
    T base_w, base_h;
    T scale_w, scale_h;
    T anchor_width, anchor_height;
    int anch_idx = i % num_anchors;
    int ar_idx = anch_idx / as_num;
    int as_idx = anch_idx % as_num;
    T aspect_ratio = aspect_ratios[ar_idx];
    T anchor_size = anchor_sizes[as_idx];
    area = stride_width * stride_height;
    area_ratios = area / aspect_ratio;
    base_w = round(sqrt(area_ratios));
    base_h = round(base_w * aspect_ratio);
    scale_w = anchor_size / stride_width;
    scale_h = anchor_size / stride_height;
    anchor_width = scale_w * base_w;
    anchor_height = scale_h * base_h;

    T xmin = (x_ctr - 0.5 * (anchor_width - 1));
    T ymin = (y_ctr - 0.5 * (anchor_height - 1));
    T xmax = (x_ctr + 0.5 * (anchor_width - 1));
    T ymax = (y_ctr + 0.5 * (anchor_height - 1));
    out[i * 4] = xmin;
    out[i * 4 + 1] = ymin;
    out[i * 4 + 2] = xmax;
    out[i * 4 + 3] = ymax;
  }
}

template <typename T>
__global__ void SetVariance(T* out, const T* var, const int vnum,
                            const int num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    out[i] = var[i % vnum];
  }
}

template <typename T>
class AnchorGeneratorOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<paddle::framework::Tensor>("Input");
    auto* anchors = ctx.Output<paddle::framework::Tensor>("Anchors");
    auto* vars = ctx.Output<paddle::framework::Tensor>("Variances");

    auto anchor_sizes = ctx.Attr<std::vector<float>>("anchor_sizes");
    auto aspect_ratios = ctx.Attr<std::vector<float>>("aspect_ratios");
    auto stride = ctx.Attr<std::vector<float>>("stride");
    auto variances = ctx.Attr<std::vector<float>>("variances");

    T offset = static_cast<T>(ctx.Attr<float>("offset"));

    auto width = input->dims()[3];
    auto height = input->dims()[2];

    int num_anchors = aspect_ratios.size() * anchor_sizes.size();

    int box_num = width * height * num_anchors;

    int block = 512;
    int grid = (box_num + block - 1) / block;

    auto stream =
        ctx.template device_context<platform::CUDADeviceContext>().stream();

    anchors->mutable_data<T>(ctx.GetPlace());
    vars->mutable_data<T>(ctx.GetPlace());

    framework::Tensor ar;
    framework::TensorFromVector(aspect_ratios, ctx.device_context(), &ar);

    framework::Tensor as;
    framework::TensorFromVector(anchor_sizes, ctx.device_context(), &as);

    framework::Tensor sd;
    framework::TensorFromVector(stride, ctx.device_context(), &sd);

    GenAnchors<T><<<grid, block, 0, stream>>>(
        anchors->data<T>(), ar.data<T>(), aspect_ratios.size(), as.data<T>(),
        anchor_sizes.size(), sd.data<T>(), stride.size(), height, width,
        offset);

    framework::Tensor v;
    framework::TensorFromVector(variances, ctx.device_context(), &v);
    grid = (box_num * 4 + block - 1) / block;
    SetVariance<T><<<grid, block, 0, stream>>>(vars->data<T>(), v.data<T>(),
                                               variances.size(), box_num * 4);
  }
};  // namespace operators

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(anchor_generator,
                        ops::AnchorGeneratorOpCUDAKernel<float>,
                        ops::AnchorGeneratorOpCUDAKernel<double>);
