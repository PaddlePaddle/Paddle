#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/memory/allocation/allocator.h>
#include <stdio.h>
#include <string>
#include <vector>
#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/detection/bbox_util.cu.h"
#include "paddle/fluid/operators/math/math_function.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

namespace {
template <typename T>
static std::pair<Tensor, Tensor> ProposalForOneImage(
    const platform::CUDADeviceContext &ctx, const Tensor &im_shape,
    const Tensor &anchors, const Tensor &variances,
    const Tensor &bbox_deltas,  // [M, 4]
    const Tensor &scores,       // [N, 1]
    int pre_nms_top_n, int post_nms_top_n, float nms_thresh, float min_size,
    float eta, bool pixel_offset) {
  // 1. pre nms
  Tensor scores_sort, index_sort;
  SortDescending<T>(ctx, scores, &scores_sort, &index_sort);
  int num = scores.numel();
  int pre_nms_num = (pre_nms_top_n <= 0 || pre_nms_top_n > num) ? scores.numel()
                                                                : pre_nms_top_n;
  scores_sort.Resize({pre_nms_num, 1});
  index_sort.Resize({pre_nms_num, 1});

  // 2. box decode and clipping
  Tensor proposals;
  proposals.mutable_data<T>({pre_nms_num, 4}, ctx.GetPlace());

  {
    platform::ForRange<platform::CUDADeviceContext> for_range(ctx, pre_nms_num);
    for_range(BoxDecodeAndClipFunctor<T>{
        anchors.data<T>(), bbox_deltas.data<T>(), variances.data<T>(),
        index_sort.data<int>(), im_shape.data<T>(), proposals.data<T>(),
        pixel_offset});
  }

  // 3. filter
  Tensor keep_index, keep_num_t;
  keep_index.mutable_data<int>({pre_nms_num}, ctx.GetPlace());
  keep_num_t.mutable_data<int>({1}, ctx.GetPlace());
  min_size = std::max(min_size, 1.0f);
  auto stream = ctx.stream();
  FilterBBoxes<T, 512><<<1, 512, 0, stream>>>(
      proposals.data<T>(), im_shape.data<T>(), min_size, pre_nms_num,
      keep_num_t.data<int>(), keep_index.data<int>(), false, pixel_offset);
  int keep_num;
  const auto gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());
  memory::Copy(platform::CPUPlace(), &keep_num, gpu_place,
               keep_num_t.data<int>(), sizeof(int), ctx.stream());
  ctx.Wait();
  keep_index.Resize({keep_num});

  Tensor scores_filter, proposals_filter;
  // Handle the case when there is no keep index left
  if (keep_num == 0) {
    math::SetConstant<platform::CUDADeviceContext, T> set_zero;
    proposals_filter.mutable_data<T>({1, 4}, ctx.GetPlace());
    scores_filter.mutable_data<T>({1, 1}, ctx.GetPlace());
    set_zero(ctx, &proposals_filter, static_cast<T>(0));
    set_zero(ctx, &scores_filter, static_cast<T>(0));
    return std::make_pair(proposals_filter, scores_filter);
  }
  proposals_filter.mutable_data<T>({keep_num, 4}, ctx.GetPlace());
  scores_filter.mutable_data<T>({keep_num, 1}, ctx.GetPlace());
  GPUGather<T>(ctx, proposals, keep_index, &proposals_filter);
  GPUGather<T>(ctx, scores_sort, keep_index, &scores_filter);

  if (nms_thresh <= 0) {
    return std::make_pair(proposals_filter, scores_filter);
  }

  // 4. nms
  Tensor keep_nms;
  NMS<T>(ctx, proposals_filter, keep_index, nms_thresh, &keep_nms,
         pixel_offset);
  if (post_nms_top_n > 0 && post_nms_top_n < keep_nms.numel()) {
    keep_nms.Resize({post_nms_top_n});
  }

  Tensor scores_nms, proposals_nms;
  proposals_nms.mutable_data<T>({keep_nms.numel(), 4}, ctx.GetPlace());
  scores_nms.mutable_data<T>({keep_nms.numel(), 1}, ctx.GetPlace());
  GPUGather<T>(ctx, proposals_filter, keep_nms, &proposals_nms);
  GPUGather<T>(ctx, scores_filter, keep_nms, &scores_nms);

  return std::make_pair(proposals_nms, scores_nms);
}
}  // namespace

template <typename DeviceContext, typename T>
class CUDAGenerateProposalsV2Kernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *scores = context.Input<Tensor>("Scores");
    auto *bbox_deltas = context.Input<Tensor>("BboxDeltas");
    auto *im_shape = context.Input<Tensor>("ImShape");
    auto anchors = GET_DATA_SAFELY(context.Input<Tensor>("Anchors"), "Input",
                                   "Anchors", "GenerateProposals");
    auto variances = GET_DATA_SAFELY(context.Input<Tensor>("Variances"),
                                     "Input", "Variances", "GenerateProposals");

    auto *rpn_rois = context.Output<LoDTensor>("RpnRois");
    auto *rpn_roi_probs = context.Output<LoDTensor>("RpnRoiProbs");

    int pre_nms_top_n = context.Attr<int>("pre_nms_topN");
    int post_nms_top_n = context.Attr<int>("post_nms_topN");
    float nms_thresh = context.Attr<float>("nms_thresh");
    float min_size = context.Attr<float>("min_size");
    float eta = context.Attr<float>("eta");
    bool pixel_offset = context.Attr<bool>("pixel_offset");
    PADDLE_ENFORCE_GE(eta, 1.,
                      platform::errors::InvalidArgument(
                          "Not support adaptive NMS. The attribute 'eta' "
                          "should not less than 1. But received eta=[%d]",
                          eta));

    auto &dev_ctx = context.template device_context<DeviceContext>();

    auto scores_dim = scores->dims();
    int64_t num = scores_dim[0];
    int64_t c_score = scores_dim[1];
    int64_t h_score = scores_dim[2];
    int64_t w_score = scores_dim[3];

    auto bbox_dim = bbox_deltas->dims();
    int64_t c_bbox = bbox_dim[1];
    int64_t h_bbox = bbox_dim[2];
    int64_t w_bbox = bbox_dim[3];

    Tensor bbox_deltas_swap, scores_swap;
    bbox_deltas_swap.mutable_data<T>({num, h_bbox, w_bbox, c_bbox},
                                     dev_ctx.GetPlace());
    scores_swap.mutable_data<T>({num, h_score, w_score, c_score},
                                dev_ctx.GetPlace());

    math::Transpose<DeviceContext, T, 4> trans;
    std::vector<int> axis = {0, 2, 3, 1};
    trans(dev_ctx, *bbox_deltas, &bbox_deltas_swap, axis);
    trans(dev_ctx, *scores, &scores_swap, axis);

    anchors.Resize({anchors.numel() / 4, 4});
    variances.Resize({variances.numel() / 4, 4});

    rpn_rois->mutable_data<T>({bbox_deltas->numel() / 4, 4},
                              context.GetPlace());
    rpn_roi_probs->mutable_data<T>({scores->numel(), 1}, context.GetPlace());

    T *rpn_rois_data = rpn_rois->data<T>();
    T *rpn_roi_probs_data = rpn_roi_probs->data<T>();

    auto place = BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace());
    auto cpu_place = platform::CPUPlace();

    int64_t num_proposals = 0;
    std::vector<size_t> offset(1, 0);
    std::vector<int> tmp_num;

    for (int64_t i = 0; i < num; ++i) {
      Tensor im_shape_slice = im_shape->Slice(i, i + 1);
      Tensor bbox_deltas_slice = bbox_deltas_swap.Slice(i, i + 1);
      Tensor scores_slice = scores_swap.Slice(i, i + 1);

      bbox_deltas_slice.Resize({h_bbox * w_bbox * c_bbox / 4, 4});
      scores_slice.Resize({h_score * w_score * c_score, 1});

      std::pair<Tensor, Tensor> box_score_pair = ProposalForOneImage<T>(
          dev_ctx, im_shape_slice, anchors, variances, bbox_deltas_slice,
          scores_slice, pre_nms_top_n, post_nms_top_n, nms_thresh, min_size,
          eta, pixel_offset);

      Tensor &proposals = box_score_pair.first;
      Tensor &scores = box_score_pair.second;

      memory::Copy(place, rpn_rois_data + num_proposals * 4, place,
                   proposals.data<T>(), sizeof(T) * proposals.numel(),
                   dev_ctx.stream());
      memory::Copy(place, rpn_roi_probs_data + num_proposals, place,
                   scores.data<T>(), sizeof(T) * scores.numel(),
                   dev_ctx.stream());
      dev_ctx.Wait();
      num_proposals += proposals.dims()[0];
      offset.emplace_back(num_proposals);
      tmp_num.push_back(proposals.dims()[0]);
    }
    if (context.HasOutput("RpnRoisNum")) {
      auto *rpn_rois_num = context.Output<Tensor>("RpnRoisNum");
      rpn_rois_num->mutable_data<int>({num}, context.GetPlace());
      int *num_data = rpn_rois_num->data<int>();
      memory::Copy(place, num_data, cpu_place, &tmp_num[0], sizeof(int) * num,
                   dev_ctx.stream());
      rpn_rois_num->Resize({num});
    }
    framework::LoD lod;
    lod.emplace_back(offset);
    rpn_rois->set_lod(lod);
    rpn_roi_probs->set_lod(lod);
    rpn_rois->Resize({num_proposals, 4});
    rpn_roi_probs->Resize({num_proposals, 1});
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(generate_proposals_v2,
                        ops::CUDAGenerateProposalsV2Kernel<
                            paddle::platform::CUDADeviceContext, float>);
