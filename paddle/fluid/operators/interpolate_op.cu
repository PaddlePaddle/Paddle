/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <algorithm>
#include <string>
#include "paddle/fluid/operators/interpolate_op.cu.h"
#include "paddle/fluid/operators/interpolate_op.h"

namespace paddle {
namespace operators {

template <typename T>
class InterpolateOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");

    auto input_dims = input->dims();
    if (input_dims.size() == 3) {  // 1D interpolation
      Interpolate1DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDAFwd<T>(ctx, *input, output);
    }
  }
};

template <typename T>
class InterpolateGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));

    auto output_grad_dims = output_grad->dims();
    if (output_grad_dims.size() == 3) {  // 1D interpolation
      Interpolate1DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDABwd<T>(ctx, input_grad, *output_grad);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(bilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(nearest_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(nearest_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(linear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(linear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(bicubic_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bicubic_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
