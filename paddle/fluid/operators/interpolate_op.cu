#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <algorithm>
#include <string>
#include "paddle/fluid/operators/interpolate_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace operators {

using framework::Tensor;
using DataLayout = framework::DataLayout;

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    if (data_layout == DataLayout::kNCHW) {
      out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    } else {
      out[tid] = in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                    in_img_idx * num_channels + channel_id];
    }
  }
}

template <typename T>
__global__ void KeNearestNeighborInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                   in_img_idy * in_img_w + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                   in_img_idx * num_channels + channel_id];
    }
    const T out_pos = out[out_id_h * output_w + out_id_w];
    platform::CudaAtomicAdd(in_pos, out_pos);
  }
}

template <typename T>
__global__ void KeLinearInterpFw(const T* in, const size_t in_img_w,
                                 const size_t input_w, T* out,
                                 const size_t out_img_w, const size_t output_h,
                                 const size_t output_w,
                                 const size_t num_channels, const float ratio_w,
                                 const bool align_corners, const int align_mode,
                                 const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;  // w
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;  // w_id

    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos =
          &in[out_id_h * out_id_w + channel_id * in_img_size + in_img_idx];
      // linear interpolation
      out[out_id_h * output_w + out_id_w] =
          w2lambda * in_pos[0] + w1lambda * in_pos[w_id];

    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idx * num_channels + channel_id];
      // linear interpolation
      out[out_id_h * output_w + out_id_w] =
          w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels];
    }
  }
}

template <typename T>
__global__ void KeLinearInterpBw(T* in, const size_t in_img_w,
                                 const size_t input_w, const T* out,
                                 const size_t out_img_w, const size_t output_h,
                                 const size_t output_w,
                                 const size_t num_channels, const T ratio_w,
                                 const bool align_corners, const int align_mode,
                                 const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idx = align_flag ? ratio_w * (out_img_idx + 0.5) - 0.5
                                : ratio_w * out_img_idx;
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;  // w
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;  // w_id

    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idx * num_channels + channel_id];
    }
    const T* out_pos = &out[out_id_w];

    if (data_layout == DataLayout::kNCHW) {
      platform::CudaAtomicAdd(&in_pos[0], w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id], w1lambda * out_pos[0]);
    } else {
      platform::CudaAtomicAdd(&in_pos[0], w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id * num_channels],
                              w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                            in_img_idy * in_img_w + in_img_idx];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                      w1lambda * in_pos[h_id * in_img_w + w_id]);
    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
              in_img_idx * num_channels + channel_id];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda *
              (w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w * num_channels] +
                      w1lambda * in_pos[h_id * in_img_w * num_channels +
                                        w_id * num_channels]);
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const T ratio_h, const T ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = align_flag ? ratio_h * (out_img_idy + 0.5) - 0.5
                                : ratio_h * out_img_idy;
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag ? ratio_w * (out_img_idx + 0.5) - 0.5
                                : ratio_w * out_img_idx;
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                   in_img_idy * in_img_w + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                   in_img_idx * num_channels + channel_id];
    }

    const T* out_pos = &out[out_id_h * output_w + out_id_w];

    if (data_layout == DataLayout::kNCHW) {
      platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id], h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[h_id * in_img_w],
                              h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[h_id * in_img_w + w_id],
                              h1lambda * w1lambda * out_pos[0]);
    } else {
      platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id * num_channels],
                              h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[h_id * in_img_w * num_channels],
                              h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos[h_id * in_img_w * num_channels + w_id * num_channels],
          h1lambda * w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpFw(
    const T* in, const size_t in_img_d, const size_t in_img_h,
    const size_t in_img_w, const size_t input_h, const size_t input_w, T* out,
    const size_t out_img_d, const size_t out_img_h, const size_t out_img_w,
    const size_t output_h, const size_t output_w, const size_t num_channels,
    const float ratio_d, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    T src_d = ratio_d * (out_img_idt + 0.5) - 0.5;
    src_d = (src_d > 0) ? src_d : 0;
    T d1lambda =
        align_flag ? src_d - in_img_idt : ratio_d * out_img_idt - in_img_idt;
    T d2lambda = 1.f - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] + w1lambda * in_pos1[w_id]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w] +
                           w1lambda * in_pos1[h_id * in_img_w + w_id])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] + w1lambda * in_pos2[w_id]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w] +
                           w1lambda * in_pos2[h_id * in_img_w + w_id]));

    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] +
                           w1lambda * in_pos1[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos1[h_id * in_img_w * num_channels +
                                              w_id * num_channels])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] +
                           w1lambda * in_pos2[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos2[h_id * in_img_w * num_channels +
                                              w_id * num_channels]));
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpBw(
    T* in, const size_t in_img_d, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, const T* out,
    const size_t out_img_d, const size_t out_img_h, const size_t out_img_w,
    const size_t output_h, const size_t output_w, const size_t num_channels,
    const T ratio_d, const T ratio_h, const T ratio_w, const bool align_corners,
    const int align_mode, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    T src_d = ratio_d * (out_img_idt + 0.5) - 0.5;
    src_d = (src_d > 0) ? src_d : 0;
    T d1lambda =
        align_flag ? src_d - in_img_idt : ratio_d * out_img_idt - in_img_idt;
    T d2lambda = 1.f - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      T* in_pos2 = &in[in_pos2_idx];

      const T* out_pos = &out[out_id_h * output_w + out_id_w];

      // trilinear interpolation grad
      platform::CudaAtomicAdd(&in_pos1[0],
                              d2lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[w_id],
                              d2lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w],
                              d2lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w + w_id],
                              d2lambda * h1lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[0],
                              d1lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[w_id],
                              d1lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w],
                              d1lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w + w_id],
                              d1lambda * h1lambda * w1lambda * out_pos[0]);
    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      T* in_pos2 = &in[in_pos2_idx];

      const T* out_pos = &out[out_id_h * output_w + out_id_w];

      // trilinear interpolation grad
      platform::CudaAtomicAdd(&in_pos1[0],
                              d2lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[w_id * num_channels],
                              d2lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w * num_channels],
                              d2lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos1[h_id * in_img_w * num_channels + w_id * num_channels],
          d2lambda * h1lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[0],
                              d1lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[w_id * num_channels],
                              d1lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w * num_channels],
                              d1lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos2[h_id * in_img_w * num_channels + w_id * num_channels],
          d1lambda * h1lambda * w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
__device__ __forceinline__ static T Kecubic_interp(const T x0, const T x1,
                                                   const T x2, const T x3,
                                                   T t) {
  T coeffs[4];
  T a = -0.75;
  T x_1 = t;
  T x_2 = 1.0 - t;
  coeffs[0] = cubic_convolution2<T>(x_1 + 1.0, a);
  coeffs[1] = cubic_convolution1<T>(x_1, a);
  coeffs[2] = cubic_convolution1<T>(x_2, a);
  coeffs[3] = cubic_convolution2<T>(x_2 + 1.0, a);
  return x0 * coeffs[0] + x1 * coeffs[1] + x2 * coeffs[2] + x3 * coeffs[3];
}

template <typename T>
__global__ void KeBicubicInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;

    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    T in_img_idy = align_corners
                       ? static_cast<T>(ratio_h * out_img_idy)
                       : static_cast<T>(ratio_h * (out_img_idy + 0.5) - 0.5);
    int input_y = floorf(in_img_idy);
    const T y_t = in_img_idy - input_y;

    T in_img_idx = align_corners
                       ? static_cast<T>(ratio_w * out_img_idx)
                       : static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    int input_x = floorf(in_img_idx);
    const T x_t = in_img_idx - input_x;

    T coefficients[4];
    const T* in_pos_0;
    const T* in_pos_1;
    const T* in_pos_2;
    const T* in_pos_3;
    int access_x_0;
    if (data_layout == DataLayout::kNCHW) {
      for (int k = 0; k < 4; k++) {
        int access_y =
            max(min(input_y - 1 + k, static_cast<int>(in_img_h - 1)), 0);
        access_x_0 = max(min(input_x - 1, static_cast<int>(in_img_w - 1)), 0);
        int access_x_1 =
            max(min(input_x + 0, static_cast<int>(in_img_w - 1)), 0);
        int access_x_2 =
            max(min(input_x + 1, static_cast<int>(in_img_w - 1)), 0);
        int access_x_3 =
            max(min(input_x + 2, static_cast<int>(in_img_w - 1)), 0);

        in_pos_0 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_0];
        in_pos_1 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_1];
        in_pos_2 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_2];
        in_pos_3 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_3];

        coefficients[k] = Kecubic_interp<T>(in_pos_0[0], in_pos_1[0],
                                            in_pos_2[0], in_pos_3[0], x_t);
      }

      out[out_id_h * output_w + out_id_w] =
          Kecubic_interp<T>(coefficients[0], coefficients[1], coefficients[2],
                            coefficients[3], y_t);

    } else {
      for (int k = 0; k < 4; k++) {
        int access_y =
            max(min(input_y - 1 + k, static_cast<int>((in_img_h - 1))), 0);
        int access_x_0 =
            max(min(input_x - 1, static_cast<int>((in_img_w - 1))), 0);
        int access_x_1 =
            max(min(input_x + 0, static_cast<int>((in_img_w - 1))), 0);
        int access_x_2 =
            max(min(input_x + 1, static_cast<int>((in_img_w - 1))), 0);
        int access_x_3 =
            max(min(input_x + 2, static_cast<int>((in_img_w - 1))), 0);

        const T* in_pos_0 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_0 * num_channels + channel_id];
        const T* in_pos_1 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_1 * num_channels + channel_id];
        const T* in_pos_2 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_2 * num_channels + channel_id];
        const T* in_pos_3 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_3 * num_channels + channel_id];

        coefficients[k] = Kecubic_interp(in_pos_0[0], in_pos_1[0], in_pos_2[0],
                                         in_pos_3[0], x_t);
      }

      out[out_id_h * output_w + out_id_w] =
          static_cast<T>(Kecubic_interp(coefficients[0], coefficients[1],
                                        coefficients[2], coefficients[3], y_t));
    }
  }
}

template <typename T>
__global__ void KeBicubicInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    T in_img_idy = align_corners
                       ? static_cast<T>(ratio_h * out_img_idy)
                       : static_cast<T>(ratio_h * (out_img_idy + 0.5) - 0.5);
    int input_y = floorf(in_img_idy);
    const T y_t = in_img_idy - input_y;

    T in_img_idx = align_corners
                       ? static_cast<T>(ratio_w * out_img_idx)
                       : static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    int input_x = floorf(in_img_idx);

    const T x_t = in_img_idx - input_x;

    T x_coeffs[4];
    T y_coeffs[4];

    get_cubic_upsample_coefficients(x_coeffs, x_t);
    get_cubic_upsample_coefficients(y_coeffs, y_t);

    const T* out_pos = &out[out_id_h * output_w + out_id_w];
    T* in_pos;

    for (int i = 0; i < 4; i++) {
      for (int j = 0; j < 4; j++) {
        int access_y = max(min(static_cast<int>(input_y - 1 + j),
                               static_cast<int>(in_img_h - 1)),
                           0);
        int access_x = max(min(static_cast<int>(input_x - 1 + i),
                               static_cast<int>(in_img_w - 1)),
                           0);
        if (data_layout == DataLayout::kNCHW) {
          in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x];
        } else {
          in_pos = &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                       access_x * num_channels + channel_id];
        }
        platform::CudaAtomicAdd(&in_pos[0],
                                (out_pos[0] * y_coeffs[j] * x_coeffs[i]));
      }
    }
  }
}

template <typename T>
static void Interpolate1DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_w = new_size[0];
  } else {
    float scale;
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      scale = scale_data[0];
    } else {
      scale = ctx.Attr<float>("scale");
    }
    if (scale > 0) {
      out_w = static_cast<int>(in_w * scale);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_w = size_data[0];
    }
  }
  PADDLE_ENFORCE_GT(out_w, 0, platform::errors::InvalidArgument(
                                  "out_w in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_w};
  } else {
    dim_out = {n, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_w = 0.f;
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1.0) / (out_w - 1.0)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_cw = c * in_w;
  int out_cw = c * out_w;
  int pixelNum = n * out_cw;

  platform::GpuLaunchConfig config =
      platform::getGpuLaunchConfig(pixelNum, ctx);

  if ("linear" == interp_method) {
    KeLinearInterpFw<T><<<config.blocks, config.threads, 0,
                          ctx.cuda_device_context().stream()>>>(
        input_data, in_w, in_cw, output_data, out_w, n, out_cw, c, ratio_w,
        align_corners, align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate2DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_h = new_size[0];
    out_w = new_size[1];
  } else {
    float scale;
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      scale = scale_data[0];
    } else {
      scale = ctx.Attr<float>("scale");
    }
    if (scale > 0) {
      out_h = static_cast<int>(in_h * scale);
      out_w = static_cast<int>(in_w * scale);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }
  }
  PADDLE_ENFORCE_GT(out_h, 0, platform::errors::InvalidArgument(
                                  "out_h in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  PADDLE_ENFORCE_GT(out_w, 0, platform::errors::InvalidArgument(
                                  "out_w in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_h, out_w};
  } else {
    dim_out = {n, out_h, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_h == out_h && in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;

  int pixelNum = n * out_chw;

  platform::GpuLaunchConfig config =
      platform::getGpuLaunchConfig(pixelNum, ctx);

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpFw<T><<<config.blocks, config.threads, 0,
                                   ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpFw<T><<<config.blocks, config.threads, 0,
                            ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, align_mode, data_layout);
  } else if ("bicubic" == interp_method) {
    KeBicubicInterpFw<
        T><<<config.blocks, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  }
}

template <typename T>
static void Interpolate3DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_d = ctx.Attr<int>("out_d");
  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  } else {
    float scale;
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      scale = scale_data[0];
    } else {
      scale = ctx.Attr<float>("scale");
    }
    if (scale > 0) {
      out_d = static_cast<int>(in_d * scale);
      out_h = static_cast<int>(in_h * scale);
      out_w = static_cast<int>(in_w * scale);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_d = size_data[0];
      out_h = size_data[1];
      out_w = size_data[2];
    }
  }
  PADDLE_ENFORCE_GT(out_d, 0, platform::errors::InvalidArgument(
                                  "out_d in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  PADDLE_ENFORCE_GT(out_h, 0, platform::errors::InvalidArgument(
                                  "out_h in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  PADDLE_ENFORCE_GT(out_w, 0, platform::errors::InvalidArgument(
                                  "out_w in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_d, out_h, out_w};
  } else {
    dim_out = {n, out_d, out_h, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(in_d) / out_d;
  }
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_dhw = in_d * in_h * in_w;
  int out_dhw = out_d * out_h * out_w;
  int in_cdhw = c * in_dhw;
  int out_cdhw = c * out_dhw;

  int pixelNum = n * out_cdhw;

  platform::GpuLaunchConfig config =
      platform::getGpuLaunchConfig(pixelNum, ctx);

  if ("trilinear" == interp_method) {
    KeTrilinearInterpFw<T><<<config.blocks, config.threads, 0,
                             ctx.cuda_device_context().stream()>>>(
        input_data, in_d, in_h, in_w, n, in_cdhw, output_data, out_d, out_h,
        out_w, n, out_cdhw, c, ratio_d, ratio_h, ratio_w, align_corners,
        align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate1DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad, const Tensor output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_w = ctx.Attr<int>("out_w");
  float scale;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    scale = scale_data[0];
  } else {
    scale = ctx.Attr<float>("scale");
  }
  if (scale > 0) {
    out_w = static_cast<int>(in_w * scale);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_w = size_data[0];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_w = new_size[0];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_w};
  } else {
    dim_grad = {n, in_w, c};
  }
  input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_w = 0.f;
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }
  int in_cw = c * in_w;
  int out_cw = c * out_w;
  int pixelNum = n * out_cw;

  platform::GpuLaunchConfig config =
      platform::getGpuLaunchConfig(pixelNum, ctx);

  if ("linear" == interp_method) {
    KeLinearInterpBw<T><<<config.blocks, config.threads, 0,
                          ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_w, in_cw, output_grad_data, out_w, n, out_cw, c,
        ratio_w, align_corners, align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate2DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad, const Tensor output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");
  float scale;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    scale = scale_data[0];
  } else {
    scale = ctx.Attr<float>("scale");
  }
  if (scale > 0) {
    out_h = static_cast<int>(in_h * scale);
    out_w = static_cast<int>(in_w * scale);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_h = size_data[0];
    out_w = size_data[1];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_h = new_size[0];
    out_w = new_size[1];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_h, in_w};
  } else {
    dim_grad = {n, in_h, in_w, c};
  }
  input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_h == out_h && in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;

  int pixelNum = n * out_chw;

  platform::GpuLaunchConfig config =
      platform::getGpuLaunchConfig(pixelNum, ctx);

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpBw<T><<<config.blocks, config.threads, 0,
                                   ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpBw<T><<<config.blocks, config.threads, 0,
                            ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, align_mode,
        data_layout);
  } else if ("bicubic" == interp_method) {
    KeBicubicInterpBw<
        T><<<config.blocks, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  }
}

template <typename T>
static void Interpolate3DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad,
                                 const Tensor& output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_d = ctx.Attr<int>("out_d");
  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");
  float scale;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    scale = scale_data[0];
  } else {
    scale = ctx.Attr<float>("scale");
  }
  if (scale > 0) {
    out_d = static_cast<int>(in_d * scale);
    out_h = static_cast<int>(in_h * scale);
    out_w = static_cast<int>(in_w * scale);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_d = size_data[0];
    out_h = size_data[1];
    out_w = size_data[2];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_d, in_h, in_w};
  } else {
    dim_grad = {n, in_d, in_h, in_w, c};
  }
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(in_d) / out_d;
  }
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_dhw = in_d * in_h * in_w;
  int out_dhw = out_d * out_h * out_w;
  int in_cdhw = c * in_dhw;
  int out_cdhw = c * out_dhw;

  int pixelNum = n * out_cdhw;

  platform::GpuLaunchConfig config =
      platform::getGpuLaunchConfig(pixelNum, ctx);

  if ("trilinear" == interp_method) {
    KeTrilinearInterpBw<T><<<config.blocks, config.threads, 0,
                             ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_d, in_h, in_w, n, in_cdhw, output_grad_data, out_d,
        out_h, out_w, n, out_cdhw, c, ratio_d, ratio_h, ratio_w, align_corners,
        align_mode, data_layout);
  }
}

template <typename T>
class InterpolateOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");

    auto input_dims = input->dims();
    if (input_dims.size() == 3) {  // 1D interpolation
      Interpolate1DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDAFwd<T>(ctx, *input, output);
    }
  }
};

template <typename T>
class InterpolateGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));

    auto output_grad_dims = output_grad->dims();
    if (output_grad_dims.size() == 3) {  // 1D interpolation
      Interpolate1DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDABwd<T>(ctx, input_grad, *output_grad);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(bilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(nearest_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(nearest_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(linear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(linear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(bicubic_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bicubic_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
