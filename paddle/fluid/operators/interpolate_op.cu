#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <string>
#include "paddle/fluid/operators/interpolate_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using framework::Tensor;

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;
    int channel_id = out_id_w / out_img_size;

    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);

    int out_img_idx = tid % out_img_w;
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                  in_img_idy * in_img_w + in_img_idx];
  }
}

template <typename T>
__global__ void KeNearestNeighborInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;
    int channel_id = out_id_w / out_img_size;

    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);

    int out_img_idx = tid % out_img_w;
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    const T out_pos = out[out_id_h * output_w + out_id_w];
    platform::CudaAtomicAdd(in_pos, out_pos);
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;
    int channel_id = out_id_w / out_img_size;

    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int out_img_idx = tid % out_img_w;
    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                          in_img_idy * in_img_w + in_img_idx];

    // bilinear interpolation
    out[out_id_h * output_w + out_id_w] =
        h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
        h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                    w1lambda * in_pos[h_id * in_img_w + w_id]);
  }
}

template <typename T>
__global__ void KeBilinearInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const T ratio_h, const T ratio_w,
    const bool align_corners, const int align_mode) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;
    int channel_id = out_id_w / out_img_size;

    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int in_img_idy = align_flag ? ratio_h * (out_img_idy + 0.5) - 0.5
                                : ratio_h * out_img_idy;
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;

    T h2lambda = 1.f - h1lambda;

    int out_img_idx = tid % out_img_w;
    int in_img_idx = align_flag ? ratio_w * (out_img_idx + 0.5) - 0.5
                                : ratio_w * out_img_idx;
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    const T* out_pos = &out[out_id_h * output_w + out_id_w];
    platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * out_pos[0]);
    platform::CudaAtomicAdd(&in_pos[w_id], h2lambda * w1lambda * out_pos[0]);
    platform::CudaAtomicAdd(&in_pos[h_id * in_img_w],
                            h1lambda * w2lambda * out_pos[0]);
    platform::CudaAtomicAdd(&in_pos[h_id * in_img_w + w_id],
                            h1lambda * w1lambda * out_pos[0]);
  }
}

template <typename T>
class InterpolateOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");
    auto* input_data = input->data<T>();

    int n = input->dims()[0];
    int c = input->dims()[1];
    int in_h = input->dims()[2];
    int in_w = input->dims()[3];

    auto interp_method = ctx.Attr<std::string>("interp_method");
    int out_h = ctx.Attr<int>("out_h");
    int out_w = ctx.Attr<int>("out_w");

    float scale = ctx.Attr<float>("scale");
    if (scale > 0) {
      out_h = in_h * scale;
      out_w = in_w * scale;
    }

    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopy(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }

    bool align_corners = ctx.Attr<bool>("align_corners");
    int align_mode = ctx.Attr<int>("align_mode");

    auto* output_data =
        output->mutable_data<T>({n, c, out_h, out_w}, ctx.GetPlace());

    int in_hw = in_h * in_w;
    int out_hw = out_h * out_w;
    int in_chw = c * in_hw;
    int out_chw = c * out_hw;

    float ratio_h = 0.f;
    float ratio_w = 0.f;
    if (out_h > 1) {
      ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                                : static_cast<float>(in_h) / out_h;
    }
    if (out_w > 1) {
      ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                                : static_cast<float>(in_w) / out_w;
    }

    if (in_h == out_h && in_w == out_w) {
      framework::TensorCopy(*input, ctx.GetPlace(), output);
      return;
    }

    int pixelNum = n * out_chw;
    int grid_dim = (pixelNum + 512 - 1) / 512;
    grid_dim = grid_dim > 8 ? 8 : grid_dim;

    if ("nearest" == interp_method) {
      KeNearestNeighborInterpFw<
          T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
          input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
          out_chw, c, ratio_h, ratio_w, align_corners);
    } else if ("bilinear" == interp_method) {
      KeBilinearInterpFw<
          T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
          input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
          out_chw, c, ratio_h, ratio_w, align_corners, align_mode);
    }
  }
};

template <typename T>
class InterpolateGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* output_grad_data = output_grad->data<T>();
    auto* input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());

    auto& device_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();
    math::SetConstant<platform::CUDADeviceContext, T> zero;
    zero(device_ctx, input_grad, static_cast<T>(0.0));

    int n = input_grad->dims()[0];
    int c = input_grad->dims()[1];
    int in_h = input_grad->dims()[2];
    int in_w = input_grad->dims()[3];

    auto interp_method = ctx.Attr<std::string>("interp_method");
    int out_h = ctx.Attr<int>("out_h");
    int out_w = ctx.Attr<int>("out_w");
    float scale = ctx.Attr<float>("scale");
    if (scale > 0) {
      out_h = in_h * scale;
      out_w = in_w * scale;
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopy(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }

    bool align_corners = ctx.Attr<bool>("align_corners");
    int align_mode = ctx.Attr<int>("align_mode");

    int in_hw = in_h * in_w;
    int out_hw = out_h * out_w;
    int in_chw = c * in_hw;
    int out_chw = c * out_hw;

    float ratio_h = 0.f;
    float ratio_w = 0.f;
    if (out_h > 1) {
      ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                                : static_cast<float>(in_h) / out_h;
    }
    if (out_w > 1) {
      ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                                : static_cast<float>(in_w) / out_w;
    }

    if (in_h == out_h && in_w == out_w) {
      framework::TensorCopy(*output_grad, ctx.GetPlace(), input_grad);
      return;
    }

    int pixelNum = n * out_chw;
    int grid_dim = (pixelNum + 512 - 1) / 512;
    grid_dim = grid_dim > 8 ? 8 : grid_dim;

    if ("nearest" == interp_method) {
      KeNearestNeighborInterpBw<
          T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
          input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h,
          out_w, n, out_chw, c, ratio_h, ratio_w, align_corners);
    } else if ("bilinear" == interp_method) {
      KeBilinearInterpBw<
          T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
          input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h,
          out_w, n, out_chw, c, ratio_h, ratio_w, align_corners, align_mode);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(bilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(nearest_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(nearest_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
