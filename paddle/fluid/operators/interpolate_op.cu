#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <string>
#include "paddle/fluid/operators/interpolate_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using framework::Tensor;
using DataLayout = framework::DataLayout;

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    if (data_layout == DataLayout::kNCHW) {
      out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    } else {
      out[tid] = in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                    in_img_idx * num_channels + channel_id];
    }
  }
}

template <typename T>
__global__ void KeNearestNeighborInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                   in_img_idy * in_img_w + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                   in_img_idx * num_channels + channel_id];
    }
    const T out_pos = out[out_id_h * output_w + out_id_w];
    platform::CudaAtomicAdd(in_pos, out_pos);
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                            in_img_idy * in_img_w + in_img_idx];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                      w1lambda * in_pos[h_id * in_img_w + w_id]);
    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
              in_img_idx * num_channels + channel_id];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda *
              (w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w * num_channels] +
                      w1lambda * in_pos[h_id * in_img_w * num_channels +
                                        w_id * num_channels]);
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const T ratio_h, const T ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = align_flag ? ratio_h * (out_img_idy + 0.5) - 0.5
                                : ratio_h * out_img_idy;
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag ? ratio_w * (out_img_idx + 0.5) - 0.5
                                : ratio_w * out_img_idx;
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                   in_img_idy * in_img_w + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                   in_img_idx * num_channels + channel_id];
    }

    const T* out_pos = &out[out_id_h * output_w + out_id_w];

    if (data_layout == DataLayout::kNCHW) {
      platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id], h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[h_id * in_img_w],
                              h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[h_id * in_img_w + w_id],
                              h1lambda * w1lambda * out_pos[0]);
    } else {
      platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id * num_channels],
                              h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[h_id * in_img_w * num_channels],
                              h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos[h_id * in_img_w * num_channels + w_id * num_channels],
          h1lambda * w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpFw(
    const T* in, const size_t in_img_d, const size_t in_img_h,
    const size_t in_img_w, const size_t input_h, const size_t input_w, T* out,
    const size_t out_img_d, const size_t out_img_h, const size_t out_img_w,
    const size_t output_h, const size_t output_w, const size_t num_channels,
    const float ratio_d, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    T src_d = ratio_d * (out_img_idt + 0.5) - 0.5;
    src_d = (src_d > 0) ? src_d : 0;
    T d1lambda =
        align_flag ? src_d - in_img_idt : ratio_d * out_img_idt - in_img_idt;
    T d2lambda = 1.f - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] + w1lambda * in_pos1[w_id]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w] +
                           w1lambda * in_pos1[h_id * in_img_w + w_id])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] + w1lambda * in_pos2[w_id]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w] +
                           w1lambda * in_pos2[h_id * in_img_w + w_id]));

    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] +
                           w1lambda * in_pos1[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos1[h_id * in_img_w * num_channels +
                                              w_id * num_channels])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] +
                           w1lambda * in_pos2[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos2[h_id * in_img_w * num_channels +
                                              w_id * num_channels]));
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpBw(
    T* in, const size_t in_img_d, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, const T* out,
    const size_t out_img_d, const size_t out_img_h, const size_t out_img_w,
    const size_t output_h, const size_t output_w, const size_t num_channels,
    const T ratio_d, const T ratio_h, const T ratio_w, const bool align_corners,
    const int align_mode, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    T src_d = ratio_d * (out_img_idt + 0.5) - 0.5;
    src_d = (src_d > 0) ? src_d : 0;
    T d1lambda =
        align_flag ? src_d - in_img_idt : ratio_d * out_img_idt - in_img_idt;
    T d2lambda = 1.f - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      T* in_pos2 = &in[in_pos2_idx];

      const T* out_pos = &out[out_id_h * output_w + out_id_w];

      // trilinear interpolation grad
      platform::CudaAtomicAdd(&in_pos1[0],
                              d2lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[w_id],
                              d2lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w],
                              d2lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w + w_id],
                              d2lambda * h1lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[0],
                              d1lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[w_id],
                              d1lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w],
                              d1lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w + w_id],
                              d1lambda * h1lambda * w1lambda * out_pos[0]);
    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      T* in_pos2 = &in[in_pos2_idx];

      const T* out_pos = &out[out_id_h * output_w + out_id_w];

      // trilinear interpolation grad
      platform::CudaAtomicAdd(&in_pos1[0],
                              d2lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[w_id * num_channels],
                              d2lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w * num_channels],
                              d2lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos1[h_id * in_img_w * num_channels + w_id * num_channels],
          d2lambda * h1lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[0],
                              d1lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[w_id * num_channels],
                              d1lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w * num_channels],
                              d1lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos2[h_id * in_img_w * num_channels + w_id * num_channels],
          d1lambda * h1lambda * w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
static void Interpolate2DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_h = new_size[0];
    out_w = new_size[1];
  } else {
    float scale;
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      scale = scale_data[0];
    } else {
      scale = ctx.Attr<float>("scale");
    }
    if (scale > 0) {
      out_h = static_cast<int>(in_h * scale);
      out_w = static_cast<int>(in_w * scale);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }
  }
  PADDLE_ENFORCE_GT(
      out_h, 0,
      "out_h in Attr(out_shape) of Op(interpolate) should be greater than 0.");
  PADDLE_ENFORCE_GT(
      out_w, 0,
      "out_w in Attr(out_shape) of Op(interpolate) should be greater than 0.");

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_h, out_w};
  } else {
    dim_out = {n, out_h, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_h == out_h && in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;

  int pixelNum = n * out_chw;
  int grid_dim = (pixelNum + 512 - 1) / 512;
  grid_dim = grid_dim > 8 ? 8 : grid_dim;

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpFw<
        T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpFw<
        T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate3DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_d = ctx.Attr<int>("out_d");
  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  } else {
    float scale;
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      scale = scale_data[0];
    } else {
      scale = ctx.Attr<float>("scale");
    }
    if (scale > 0) {
      out_d = static_cast<int>(in_d * scale);
      out_h = static_cast<int>(in_h * scale);
      out_w = static_cast<int>(in_w * scale);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_d = size_data[0];
      out_h = size_data[1];
      out_w = size_data[2];
    }
  }
  PADDLE_ENFORCE_GT(
      out_d, 0,
      "out_d in Attr(out_shape) of Op(interpolate) should be greater than 0.");
  PADDLE_ENFORCE_GT(
      out_h, 0,
      "out_h in Attr(out_shape) of Op(interpolate) should be greater than 0.");
  PADDLE_ENFORCE_GT(
      out_w, 0,
      "out_w in Attr(out_shape) of Op(interpolate) should be greater than 0.");

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_d, out_h, out_w};
  } else {
    dim_out = {n, out_d, out_h, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(in_d) / out_d;
  }
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_dhw = in_d * in_h * in_w;
  int out_dhw = out_d * out_h * out_w;
  int in_cdhw = c * in_dhw;
  int out_cdhw = c * out_dhw;

  int pixelNum = n * out_cdhw;
  int grid_dim = (pixelNum + 512 - 1) / 512;
  grid_dim = grid_dim > 8 ? 8 : grid_dim;

  if ("trilinear" == interp_method) {
    KeTrilinearInterpFw<
        T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_data, in_d, in_h, in_w, n, in_cdhw, output_data, out_d, out_h,
        out_w, n, out_cdhw, c, ratio_d, ratio_h, ratio_w, align_corners,
        align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate2DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad, const Tensor output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");
  float scale;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    scale = scale_data[0];
  } else {
    scale = ctx.Attr<float>("scale");
  }
  if (scale > 0) {
    out_h = static_cast<int>(in_h * scale);
    out_w = static_cast<int>(in_w * scale);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_h = size_data[0];
    out_w = size_data[1];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_h = new_size[0];
    out_w = new_size[1];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_h, in_w};
  } else {
    dim_grad = {n, in_h, in_w, c};
  }
  input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_h == out_h && in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;

  int pixelNum = n * out_chw;
  int grid_dim = (pixelNum + 512 - 1) / 512;
  grid_dim = grid_dim > 8 ? 8 : grid_dim;

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpBw<
        T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpBw<
        T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, align_mode,
        data_layout);
  }
}

template <typename T>
static void Interpolate3DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad,
                                 const Tensor& output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_d = ctx.Attr<int>("out_d");
  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");
  float scale;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    scale = scale_data[0];
  } else {
    scale = ctx.Attr<float>("scale");
  }
  if (scale > 0) {
    out_d = static_cast<int>(in_d * scale);
    out_h = static_cast<int>(in_h * scale);
    out_w = static_cast<int>(in_w * scale);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_d = size_data[0];
    out_h = size_data[1];
    out_w = size_data[2];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_d, in_h, in_w};
  } else {
    dim_grad = {n, in_d, in_h, in_w, c};
  }
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(in_d) / out_d;
  }
  if (out_h > 1) {
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(in_h) / out_h;
  }
  if (out_w > 1) {
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(in_w) / out_w;
  }

  int in_dhw = in_d * in_h * in_w;
  int out_dhw = out_d * out_h * out_w;
  int in_cdhw = c * in_dhw;
  int out_cdhw = c * out_dhw;

  int pixelNum = n * out_cdhw;
  int grid_dim = (pixelNum + 512 - 1) / 512;
  grid_dim = grid_dim > 8 ? 8 : grid_dim;

  if ("trilinear" == interp_method) {
    KeTrilinearInterpBw<
        T><<<grid_dim, 512, 0, ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_d, in_h, in_w, n, in_cdhw, output_grad_data, out_d,
        out_h, out_w, n, out_cdhw, c, ratio_d, ratio_h, ratio_w, align_corners,
        align_mode, data_layout);
  }
}

template <typename T>
class InterpolateOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");

    auto input_dims = input->dims();
    if (input_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDAFwd<T>(ctx, *input, output);
    }
  }
};

template <typename T>
class InterpolateGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));

    auto output_grad_dims = output_grad->dims();
    if (output_grad_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDABwd<T>(ctx, input_grad, *output_grad);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(bilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(nearest_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(nearest_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp, ops::InterpolateOpCUDAKernel<float>,
                        ops::InterpolateOpCUDAKernel<double>,
                        ops::InterpolateOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp_grad,
                        ops::InterpolateGradOpCUDAKernel<float>,
                        ops::InterpolateGradOpCUDAKernel<double>);
