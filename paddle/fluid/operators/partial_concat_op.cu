#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/platform/device_context.h>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/partial_concat_op.h"
#include "paddle/fluid/platform/float16.h"

namespace plat = paddle::platform;

namespace paddle {
namespace operators {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

using LoDTensor = framework::LoDTensor;
using Tensor = framework::Tensor;

template <class T>
__global__ void ConcatPartialCUDAKernel(T **in, T *out, int64_t all_length,
                                        int64_t in_batch_len,
                                        int64_t start_index,
                                        int64_t out_batch_len,
                                        int64_t part_length) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < all_length) {
    int64_t bs_id = id / out_batch_len;
    int64_t bs_index = id % out_batch_len;
    int64_t var_id = bs_index / part_length;
    int64_t part_index = bs_index % part_length;
    int64_t in_id = start_index + part_index;
    const T *tmp = in[var_id];
    out[id] = tmp[bs_id * in_batch_len + in_id];
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void ConcatPartialGradCUDAKernel(
    T **in, const T *out, int64_t all_length, int64_t in_batch_len,
    int64_t start_index, int64_t out_batch_len, int64_t part_length) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < all_length) {
    int64_t bs_id = id / out_batch_len;
    int64_t bs_index = id % out_batch_len;
    int64_t var_id = bs_index / part_length;
    int64_t part_index = bs_index % part_length;
    int64_t in_id = start_index + part_index;
    T *tmp = in[var_id];
    tmp[bs_id * in_batch_len + in_id] = out[id];
    id += blockDim.x * gridDim.x;
  }
}

template <typename T>
class PartialConcatOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto in_vars = ctx.MultiInput<Tensor>("X");
    Tensor *out = ctx.Output<Tensor>("Out");
    PADDLE_ENFORCE_EQ(in_vars[0] != nullptr, true,
                      platform::errors::InvalidArgument(
                          "The input of partial concat should not be null."));

    auto input_dim = in_vars[0]->dims();
    PADDLE_ENFORCE_EQ(input_dim.size(), 2,
                      platform::errors::InvalidArgument(
                          "Only supports 2-D array with batch size in the 1st "
                          "dimension and data in the 2nd."));
    auto in_size = input_dim[1];
    // may be negative
    auto start_index = ctx.Attr<int>("start_index");
    start_index = ComputeStartIndex(start_index, in_size);

    auto partial_len = ctx.Attr<int>("length");
    if (partial_len < 0) {
      partial_len = in_size - start_index;
    }

    int in_num = in_vars.size();
    int batch_size = input_dim[0];
    int out_batch_len = partial_len * in_num;
    int all_length = batch_size * out_batch_len;

    constexpr size_t theory_sm_threads = 1024;
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    auto sm_count = max_threads / theory_sm_threads;
    size_t tile_size = 0;
    int grids;
    int blocks;
    auto ComputeKernelParameter = [&](size_t length) {
      if (length >= max_threads)
        tile_size = 1024;
      else if (length < max_threads && length > sm_count * 128)
        tile_size = 512;
      else if (length <= sm_count * 128)
        tile_size = 256;
      grids = CEIL_DIV(length, tile_size);
      blocks = tile_size;
    };

    auto place = ctx.GetPlace();
    T *out_data = out->mutable_data<T>(place);

    std::vector<const T *> in_data;
    for (int i = 0; i < in_num; ++i)
      in_data.emplace_back(in_vars[i]->data<T>());

    auto tmp_in_array = memory::Alloc(dev_ctx, in_data.size() * sizeof(T *));
    memory::Copy(BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                 tmp_in_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(in_data.data()),
                 in_data.size() * sizeof(T *), dev_ctx.stream());

    T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());
    ComputeKernelParameter(all_length);
    ConcatPartialCUDAKernel<T><<<grids, blocks, 0, stream>>>(
        in_array_data, out->data<T>(), all_length, in_size, start_index,
        out_batch_len, partial_len);
  }
};

template <typename T>
class PartialConcatGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *out_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto ins = ctx.MultiInput<LoDTensor>("X");
    auto outs = ctx.MultiOutput<LoDTensor>(framework::GradVarName("X"));

    PADDLE_ENFORCE_EQ(ins[0] != nullptr, true,
                      platform::errors::InvalidArgument(
                          "The input of partial concat should not be null."));
    // all parameters
    auto batch_size = ins[0]->dims()[0];
    auto in_size = ins[0]->dims()[1];
    // may be negative
    auto start_index = ctx.Attr<int>("start_index");
    start_index = ComputeStartIndex(start_index, in_size);
    auto partial_len = ctx.Attr<int>("length");
    if (partial_len < 0) partial_len = in_size - start_index;

    auto in_num = ins.size();
    auto grad_batch_len = partial_len * in_num;
    auto all_length = grad_batch_len * batch_size;
    // initialize
    auto &place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();
    for (size_t i = 0; i < outs.size(); ++i) {
      outs[i]->mutable_data<T>(ctx.GetPlace());
      auto dxt = framework::EigenVector<T>::Flatten(*outs[i]);
      dxt.device(place) = dxt.constant(static_cast<T>(0));
    }

    constexpr size_t theory_sm_threads = 1024;
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    auto sm_count = max_threads / theory_sm_threads;
    size_t tile_size = 0;
    int grids;
    int blocks;
    auto ComputeKernelParameter = [&](size_t length) {
      if (length >= max_threads)
        tile_size = 1024;
      else if (length < max_threads && length > sm_count * 128)
        tile_size = 512;
      else if (length <= sm_count * 128)
        tile_size = 256;
      grids = CEIL_DIV(length, tile_size);
      blocks = tile_size;
    };

    std::vector<const T *> out_data;
    for (size_t i = 0; i < in_num; ++i) {
      out_data.emplace_back(outs[i]->data<T>());
    }
    auto tmp_out_array = memory::Alloc(dev_ctx, out_data.size() * sizeof(T *));

    memory::Copy(BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace()),
                 tmp_out_array->ptr(), platform::CPUPlace(),
                 reinterpret_cast<void *>(out_data.data()),
                 out_data.size() * sizeof(T *), dev_ctx.stream());

    T **out_grad_data = reinterpret_cast<T **>(tmp_out_array->ptr());
    ComputeKernelParameter(all_length);
    ConcatPartialGradCUDAKernel<T><<<grids, blocks, 0, stream>>>(
        out_grad_data, out_grad->data<T>(), all_length, in_size, start_index,
        grad_batch_len, partial_len);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(partial_concat, ops::PartialConcatOpCUDAKernel<float>,
                        ops::PartialConcatOpCUDAKernel<double>,
                        ops::PartialConcatOpCUDAKernel<int>,
                        ops::PartialConcatOpCUDAKernel<int64_t>,
                        ops::PartialConcatOpCUDAKernel<plat::float16>);

REGISTER_OP_CUDA_KERNEL(partial_concat_grad,
                        ops::PartialConcatGradOpCUDAKernel<float>,
                        ops::PartialConcatGradOpCUDAKernel<double>,
                        ops::PartialConcatGradOpCUDAKernel<int>,
                        ops::PartialConcatGradOpCUDAKernel<int64_t>,
                        ops::PartialConcatGradOpCUDAKernel<plat::float16>);
