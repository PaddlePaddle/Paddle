#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/roi_align_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <class T>
__device__ T BilinearInterpolate(const T* input_data, const int height,
                                 const int width, T y, T x) {
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    return 0;
  }
  y = y <= 0 ? 0 : y;
  x = x <= 0 ? 0 : x;
  int y_low = static_cast<int>(y);
  int x_low = static_cast<int>(x);
  int y_high;
  int x_high;
  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = static_cast<T>(y_low);
  } else {
    y_high = y_low + 1;
  }
  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = static_cast<T>(x_low);
  } else {
    x_high = x_low + 1;
  }
  T ly = y - y_low, lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  T v1 = input_data[y_low * width + x_low];
  T v2 = input_data[y_low * width + x_high];
  T v3 = input_data[y_high * width + x_low];
  T v4 = input_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <class T>
__device__ void BilinearInterpolateGradient(const int height, const int width,
                                            T y, T x, T* w1, T* w2, T* w3,
                                            T* w4, int* x_low, int* x_high,
                                            int* y_low, int* y_high) {
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    return;
  }

  y = y <= 0 ? 0 : y;
  x = x <= 0 ? 0 : x;
  *y_low = static_cast<int>(y);
  *x_low = static_cast<int>(x);
  if (*y_low >= height - 1) {
    *y_high = *y_low = height - 1;
    y = static_cast<T>(*y_low);
  } else {
    *y_high = *y_low + 1;
  }
  if (*x_low >= width - 1) {
    *x_high = *x_low = width - 1;
    x = static_cast<T>(*x_low);
  } else {
    *x_high = *x_low + 1;
  }
  T ly = y - *y_low, lx = x - *x_low;
  T hy = 1. - ly, hx = 1. - lx;
  *w1 = hy * hx, *w2 = hy * lx, *w3 = ly * hx, *w4 = ly * lx;

  return;
}

template <class T>
__global__ void GPUROIAlignForward(
    const int nthreads, const T* input_data, const T* input_rois,
    const float spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int sampling_ratio, int* roi_batch_id_data, T* output_data) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % channels;
    int n = i / pooled_width / pooled_height / channels;

    const T* offset_input_rois = input_rois + n * kROISize;
    int roi_batch_ind = roi_batch_id_data[n];

    T roi_xmin = offset_input_rois[0] * spatial_scale;
    T roi_ymin = offset_input_rois[1] * spatial_scale;
    T roi_xmax = offset_input_rois[2] * spatial_scale;
    T roi_ymax = offset_input_rois[3] * spatial_scale;

    T roi_width = max(roi_xmax - roi_xmin, static_cast<T>(1.));
    T roi_height = max(roi_ymax - roi_ymin, static_cast<T>(1.));
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_input_data =
        input_data + (roi_batch_ind * channels + c) * height * width;

    int roi_bin_grid_h = (sampling_ratio > 0)
                             ? sampling_ratio
                             : ceil(roi_height / pooled_height);
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);
    const T count = roi_bin_grid_h * roi_bin_grid_w;
    T output_val = 0;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) {
      const T y = roi_ymin + ph * bin_size_h +
                  static_cast<T>(iy + .5f) * bin_size_h /
                      static_cast<T>(roi_bin_grid_h);
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_xmin + pw * bin_size_w +
                    static_cast<T>(ix + .5f) * bin_size_w /
                        static_cast<T>(roi_bin_grid_w);
        T val = BilinearInterpolate(offset_input_data, height, width, y, x);
        output_val += val;
      }
    }
    output_val /= count;
    output_data[i] = output_val;
  }
}

template <typename T>
__global__ void GPUROIAlignBackward(const int nthreads, const T* input_rois,
                                    const T* out_grad, const int num_rois,
                                    const float spatial_scale,
                                    const int channels, const int height,
                                    const int width, const int pooled_height,
                                    const int pooled_width,
                                    const int sampling_ratio,
                                    int* roi_batch_id_data, T* input_grad) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % channels;
    int n = i / pooled_width / pooled_height / channels;
    const T* offset_input_rois = input_rois + n * kROISize;
    int roi_batch_ind = roi_batch_id_data[n];

    T roi_xmin = offset_input_rois[0] * spatial_scale;
    T roi_ymin = offset_input_rois[1] * spatial_scale;
    T roi_xmax = offset_input_rois[2] * spatial_scale;
    T roi_ymax = offset_input_rois[3] * spatial_scale;

    T roi_width = max(roi_xmax - roi_xmin, static_cast<T>(1.));
    T roi_height = max(roi_ymax - roi_ymin, static_cast<T>(1.));
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_input_grad =
        input_grad + (roi_batch_ind * channels + c) * height * width;

    const T* offset_out_grad =
        out_grad + (n * channels + c) * pooled_height * pooled_width;
    const T out_grad_this_bin = offset_out_grad[ph * pooled_width + pw];

    int roi_bin_grid_h = (sampling_ratio > 0)
                             ? sampling_ratio
                             : ceil(roi_height / pooled_height);
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    const T count = roi_bin_grid_h * roi_bin_grid_w;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) {
      const T y = roi_ymin + ph * bin_size_h +
                  static_cast<T>(iy + .5f) * bin_size_h /
                      static_cast<T>(roi_bin_grid_h);
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_xmin + pw * bin_size_w +
                    static_cast<T>(ix + .5f) * bin_size_w /
                        static_cast<T>(roi_bin_grid_w);
        T w1 = 0, w2 = 0, w3 = 0, w4 = 0;
        int x_low = -1, x_high = -1, y_low = -1, y_high = -1;
        BilinearInterpolateGradient(height, width, y, x, &w1, &w2, &w3, &w4,
                                    &x_low, &x_high, &y_low, &y_high);
        T diff1 = out_grad_this_bin * w1 / count;
        T diff2 = out_grad_this_bin * w2 / count;
        T diff3 = out_grad_this_bin * w3 / count;
        T diff4 = out_grad_this_bin * w4 / count;
        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          platform::CudaAtomicAdd(offset_input_grad + y_low * width + x_low,
                                  diff1);
          platform::CudaAtomicAdd(offset_input_grad + y_low * width + x_high,
                                  diff2);
          platform::CudaAtomicAdd(offset_input_grad + y_high * width + x_low,
                                  diff3);
          platform::CudaAtomicAdd(offset_input_grad + y_high * width + x_high,
                                  diff4);
        }
      }
    }
  }
}

template <typename Place, typename T>
class GPUROIAlignOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");
    auto* out = ctx.Output<Tensor>("Out");

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");
    auto sampling_ratio = ctx.Attr<int>("sampling_ratio");

    auto in_dims = in->dims();
    int batch_size = in_dims[0];
    int channels = in_dims[1];
    int height = in_dims[2];
    int width = in_dims[3];

    int rois_num = rois->dims()[0];

    if (rois_num == 0) return;

    int output_size = out->numel();
    int blocks = NumBlocks(output_size);
    int threads = kNumCUDAThreads;

    Tensor roi_batch_id_list;
    roi_batch_id_list.Resize({rois_num});
    auto cplace = platform::CPUPlace();
    int* roi_batch_id_data = roi_batch_id_list.mutable_data<int>(cplace);
    auto rois_lod = rois->lod().back();
    int rois_batch_size = rois_lod.size() - 1;
    PADDLE_ENFORCE_EQ(
        rois_batch_size, batch_size,
        "The rois_batch_size and imgs batch_size must be the same.");
    int rois_num_with_lod = rois_lod[rois_batch_size];
    PADDLE_ENFORCE_EQ(rois_num, rois_num_with_lod,
                      "The rois_num from input and lod must be the same.");
    for (int n = 0; n < rois_batch_size; ++n) {
      for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
        roi_batch_id_data[i] = n;
      }
    }
    auto& dev_ctx = ctx.cuda_device_context();
    auto& allocator =
        platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx);
    int bytes = roi_batch_id_list.numel() * sizeof(int);
    auto roi_ptr = allocator.Allocate(bytes);
    int* roi_id_data = reinterpret_cast<int*>(roi_ptr->ptr());
    const auto gplace = boost::get<platform::CUDAPlace>(ctx.GetPlace());
    memory::Copy(gplace, roi_id_data, cplace, roi_batch_id_data, bytes,
                 dev_ctx.stream());
    GPUROIAlignForward<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
        output_size, in->data<T>(), rois->data<T>(), spatial_scale, channels,
        height, width, pooled_height, pooled_width, sampling_ratio, roi_id_data,
        out->mutable_data<T>(ctx.GetPlace()));
  }
};

template <typename Place, typename T>
class GPUROIAlignGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<Tensor>("X");
    auto* rois = ctx.Input<LoDTensor>("ROIs");

    auto* out_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* in_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto pooled_height = ctx.Attr<int>("pooled_height");
    auto pooled_width = ctx.Attr<int>("pooled_width");
    auto spatial_scale = ctx.Attr<float>("spatial_scale");
    auto sampling_ratio = ctx.Attr<int>("sampling_ratio");

    int rois_num = rois->dims()[0];
    int channels = in->dims()[1];
    int height = in->dims()[2];
    int width = in->dims()[3];

    if (!in_grad) {
      return;
    }
    Tensor roi_batch_id_list;
    roi_batch_id_list.Resize({rois_num});
    auto cplace = platform::CPUPlace();
    int* roi_batch_id_data = roi_batch_id_list.mutable_data<int>(cplace);
    auto rois_lod = rois->lod().back();
    int rois_batch_size = rois_lod.size() - 1;
    for (int n = 0; n < rois_batch_size; ++n) {
      for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
        roi_batch_id_data[i] = n;
      }
    }
    auto& dev_ctx = ctx.cuda_device_context();
    auto& allocator =
        platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx);
    auto roi_ptr = allocator.Allocate(roi_batch_id_list.numel() * sizeof(int));
    int* roi_id_data = reinterpret_cast<int*>(roi_ptr->ptr());
    int bytes = roi_batch_id_list.numel() * sizeof(int);
    const auto gplace = boost::get<platform::CUDAPlace>(ctx.GetPlace());
    memory::Copy(gplace, roi_id_data, cplace, roi_batch_id_data, bytes,
                 dev_ctx.stream());
    in_grad->mutable_data<T>(ctx.GetPlace());
    math::SetConstant<Place, T> set_zero;
    set_zero(dev_ctx, in_grad, static_cast<T>(0));

    int output_grad_size = out_grad->numel();
    int blocks = NumBlocks(output_grad_size);
    int threads = kNumCUDAThreads;

    if (output_grad_size > 0) {
      GPUROIAlignBackward<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
          output_grad_size, rois->data<T>(), out_grad->data<T>(), rois_num,
          spatial_scale, channels, height, width, pooled_height, pooled_width,
          sampling_ratio, roi_id_data,
          in_grad->mutable_data<T>(ctx.GetPlace()));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roi_align,
    ops::GPUROIAlignOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUROIAlignOpKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    roi_align_grad,
    ops::GPUROIAlignGradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::GPUROIAlignGradOpKernel<paddle::platform::CUDADeviceContext, double>);
