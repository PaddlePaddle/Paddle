#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/momentum_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void MomentumKernel(const T* p, const T* g, const T* v,
                               const T* learning_rate, const T mu,
                               const int64_t num, bool use_nesterov, T* p_out,
                               T* v_out) {
  T lr = learning_rate[0];
  if (use_nesterov) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
         i += blockDim.x * gridDim.x) {
      T g_val = g[i];
      T v_new = v[i] * mu + g_val;
      v_out[i] = v_new;
      p_out[i] = p[i] - (g_val - v_new * mu) * lr;
    }
  } else {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
         i += blockDim.x * gridDim.x) {
      T v_new = v[i] * mu + g[i];
      v_out[i] = v_new;
      p_out[i] = p[i] - lr * v_new;
    }
  }
}

template <typename T>
class MomentumOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto param_out = ctx.Output<framework::Tensor>("ParamOut");
    auto velocity_out = ctx.Output<framework::Tensor>("VelocityOut");
    auto param = ctx.Input<framework::Tensor>("Param");
    auto velocity = ctx.Input<framework::Tensor>("Velocity");
    auto grad = ctx.Input<framework::Tensor>("Grad");
    auto learning_rate = ctx.Input<framework::Tensor>("LearningRate");

    T* p_out = param_out->mutable_data<T>(ctx.GetPlace());
    T* v_out = velocity_out->mutable_data<T>(ctx.GetPlace());

    T mu = static_cast<T>(ctx.Attr<float>("mu"));
    bool use_nesterov = ctx.Attr<bool>("use_nesterov");

    auto* p = param->data<T>();
    auto* v = velocity->data<T>();
    auto* g = grad->data<T>();
    auto* lr = learning_rate->data<T>();

    int block = 512;
    int grid = (param->numel() + block - 1) / block;
    MomentumKernel<T><<<grid, block, 0, ctx.cuda_device_context().stream()>>>(
        p, g, v, lr, mu, param->numel(), use_nesterov, p_out, v_out);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(momentum, ops::MomentumOpCUDAKernel<float>,
                        ops::MomentumOpCUDAKernel<double>);
