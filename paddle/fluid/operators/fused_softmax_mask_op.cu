#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
// this file is inspired by:
// https://github.com/NVIDIA/Megatron-LM/blob/main/megatron/fused_kernels/scaled_masked_softmax.h

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#endif
#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
#endif
#include <stdint.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>
#include <algorithm>
#include <string>

#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/fused_softmax_mask_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using framework::Tensor;

#ifdef PADDLE_WITH_HIP
#define WARP_SIZE 64
#else
#define WARP_SIZE 32
#endif

#define MASK 0xffffffff

namespace plat = paddle::platform;

__device__ __inline__ void load_data(plat::float16* dst,
                                     const plat::float16* src) {
  *(reinterpret_cast<float2*>(dst)) = *(reinterpret_cast<const float2*>(src));
}

__device__ __inline__ void load_data(float* dst, const float* src) {
  *(reinterpret_cast<float4*>(dst)) = *(reinterpret_cast<const float4*>(src));
}

int get_pow2(int value) {
  // get next pow2 index
  int pow2_index = 0;
  while ((1 << pow2_index) < value) {
    ++pow2_index;
  }
  return pow2_index;
}

template <typename T>
struct AddOP {
  __device__ __forceinline__ T operator()(T a, T b) const { return a + b; }
};

template <typename T>
struct MaxOP {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};

template <typename T>
__device__ __forceinline__ T warp_shfl_xor(T value, int laneMask, int width,
                                           unsigned int mask = MASK) {
#if CUDA_VERSION >= 9000
  return __shfl_xor_sync(mask, value, laneMask, width);
#else
  return __shfl_xor(value, laneMask, width);
#endif
}

template <typename T, int batch, int width, template <typename> class ReduceOp>
__device__ __forceinline__ void warp_reduce(T* sum) {
  ReduceOp<T> r;
#pragma unroll
  for (int offset = width / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < batch; ++i) {
      T b = warp_shfl_xor(sum[i], offset, width);
      sum[i] = r(sum[i], b);
    }
  }
}

// T == fp16
template <typename T, int pow2_index>
__global__ void SoftmaxMaskFuseGPUKernel(const T* x_data, const T* mask_data,
                                         T* y_data, int batch_count,
                                         int key_seq_len) {
  // the forward gpu kernel
  constexpr int next_pow2 = 1 << pow2_index;
  constexpr int warp_size = (next_pow2 < WARP_SIZE) ? next_pow2 : WARP_SIZE;
  constexpr int kLocalIterations = std::max(next_pow2 / warp_size, 4);
  constexpr int kLocalBatchSize = (next_pow2 <= 128) ? 2 : 1;
  constexpr int kOneLoadingCounts = 4;

  int data_first_idx =
      (blockDim.y *
           (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z)) +
       threadIdx.y) *
      kLocalBatchSize;

  int mask_fist_idx =
      (blockDim.y * (blockIdx.x + gridDim.x * blockIdx.z) + threadIdx.y) *
      kLocalBatchSize;

  // batch_count might not be a multiple of kLocalBatchSize. Check how
  // many batches have to computed within this WARP.
  int local_batches = batch_count - data_first_idx;
  if (local_batches > kLocalBatchSize) local_batches = kLocalBatchSize;

  // might be many batches per warp. compute the index within the batch
  int local_idx = threadIdx.x;

  int x_offset = data_first_idx * key_seq_len + kOneLoadingCounts * local_idx;
  int mask_offset = mask_fist_idx * key_seq_len + kOneLoadingCounts * local_idx;
  x_data += x_offset;
  mask_data += mask_offset;
  y_data += x_offset;

  // using float for all inter compute
  float data[kLocalBatchSize][kLocalIterations];
  T temp_data[kOneLoadingCounts];
  T temp_mask[kOneLoadingCounts];

#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
    int batch_data = (i >= local_batches) ? 0 : key_seq_len;

#pragma unroll
    for (int ii = 0; ii < kLocalIterations; ii += kOneLoadingCounts) {
      int data_index = kOneLoadingCounts * local_idx + ii * warp_size;

      if (data_index < batch_data) {
        int itr_idx = i * key_seq_len + ii * warp_size;

        // efficiently load data from global memory
        load_data(temp_data, x_data + itr_idx);
        load_data(temp_mask, mask_data + itr_idx);

#pragma unroll
        for (int counter = 0; counter < kOneLoadingCounts; ++counter) {
          data[i][ii + counter] = static_cast<float>(temp_data[counter]) +
                                  static_cast<float>(temp_mask[counter]);
        }
      } else {
#pragma unroll
        for (int counter = 0; counter < kOneLoadingCounts; ++counter) {
          data[i][ii + counter] = -std::numeric_limits<float>::infinity();
        }
      }
    }
  }

  // compute max_value
  // max value for each batch for current warp
  float samples_max_value[kLocalBatchSize];
#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
    samples_max_value[i] = data[i][0];
#pragma unroll
    for (int ii = 1; ii < kLocalIterations; ++ii) {
      samples_max_value[i] = (samples_max_value[i] > data[i][ii])
                                 ? samples_max_value[i]
                                 : data[i][ii];
    }
  }
  // max value for each batch for all warp
  warp_reduce<float, kLocalBatchSize, warp_size, MaxOP>(samples_max_value);

  // compute the sum for each batch for current warp
  float samples_sum[kLocalBatchSize]{0.0f};
#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
#pragma unroll
    for (int ii = 0; ii < kLocalIterations; ++ii) {
      data[i][ii] = std::exp((data[i][ii] - samples_max_value[i]));
      samples_sum[i] += data[i][ii];
    }
  }
  // samples_sum for each batch for all warp
  warp_reduce<float, kLocalBatchSize, warp_size, AddOP>(samples_sum);

  // load the result from device back to host
  T samples_out[kOneLoadingCounts];
#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int ii = 0; ii < kLocalIterations; ii += kOneLoadingCounts) {
      int idx = kOneLoadingCounts * local_idx + ii * warp_size;
      if (idx < key_seq_len) {
#pragma unroll
        for (int counter = 0; counter < kOneLoadingCounts; ++counter) {
          samples_out[counter] = data[i][ii + counter] / samples_sum[i];
        }
        load_data(y_data + i * key_seq_len + ii * warp_size, samples_out);
      } else {
        break;
      }
    }
  }
}

template <typename T, int pow2_index>
__global__ void SoftmaxMaskFuseGradGPUKernel(const T* grad_input,
                                             T* grad_output,
                                             const T* softmax_rst,
                                             int batch_count, int key_seq_len) {
  constexpr int next_pow2 = 1 << pow2_index;
  constexpr int warp_size = (next_pow2 < WARP_SIZE) ? next_pow2 : WARP_SIZE;
  constexpr int kLocalIterations = std::max(next_pow2 / warp_size, 4);
  constexpr int kLocalBatchSize = (next_pow2 <= 128) ? 2 : 1;
  constexpr int kOneLoadingCounts = 4;

  int data_first_idx =
      (blockDim.y * blockIdx.x + threadIdx.y) * kLocalBatchSize;

  // batch_count might not be a multiple of kLocalBatchSize. Check how
  // many batches have to computed within this WARP.
  int local_batches = batch_count - data_first_idx;
  if (local_batches > kLocalBatchSize) local_batches = kLocalBatchSize;

  // might be many batches per warp. compute the index within the batch
  int local_idx = threadIdx.x;

  // the first element to process by the current thread
  int offset = data_first_idx * key_seq_len + kOneLoadingCounts * local_idx;
  grad_input += offset;
  grad_output += offset;
  softmax_rst += offset;

  // using float for all inter compute
  float grad_input_reg[kLocalBatchSize][kLocalIterations]{0.0f};
  float softmax_rst_reg[kLocalBatchSize][kLocalIterations]{0.0f};
  T temp_grad_input[kOneLoadingCounts];
  T temp_softmax_rst[kOneLoadingCounts];

#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
    int batch_data = (i >= local_batches) ? 0 : key_seq_len;

#pragma unroll
    for (int ii = 0; ii < kLocalIterations; ii += kOneLoadingCounts) {
      int data_index = kOneLoadingCounts * local_idx + ii * WARP_SIZE;
      if (data_index < batch_data) {
        load_data(temp_grad_input,
                  grad_input + i * key_seq_len + ii * warp_size);
        load_data(temp_softmax_rst,
                  softmax_rst + i * key_seq_len + ii * warp_size);

#pragma unroll
        for (int counter = 0; counter < kOneLoadingCounts; ++counter) {
          softmax_rst_reg[i][ii + counter] =
              static_cast<float>(temp_softmax_rst[counter]);
        }
#pragma unroll
        for (int counter = 0; counter < kOneLoadingCounts; ++counter) {
          grad_input_reg[i][ii + counter] =
              static_cast<float>(temp_grad_input[counter]) *
              softmax_rst_reg[i][ii + counter];
        }
      }
    }
  }

  float samples_sum[kLocalBatchSize];
#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
    samples_sum[i] = grad_input_reg[i][0];
#pragma unroll
    for (int ii = 1; ii < kLocalIterations; ++ii) {
      samples_sum[i] += grad_input_reg[i][ii];
    }
  }
  warp_reduce<float, kLocalBatchSize, warp_size, AddOP>(samples_sum);

#pragma unroll
  for (int i = 0; i < kLocalBatchSize; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int ii = 0; ii < kLocalIterations; ii += kOneLoadingCounts) {
      int data_index = kOneLoadingCounts * local_idx + ii * warp_size;
      if (data_index < key_seq_len) {
        // compute gradients
        T samples_out[kOneLoadingCounts];
#pragma unroll
        for (int counter = 0; counter < kOneLoadingCounts; ++counter) {
          samples_out[counter] =
              grad_input_reg[i][ii + counter] -
              softmax_rst_reg[i][ii + counter] * samples_sum[i];
        }
        load_data(grad_output + i * key_seq_len + ii * warp_size, samples_out);
      }
    }
  }
}

// T only supports fp16
// leave as template only for future update
template <typename Place, typename T>
class SoftmaxMaskFuseKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* mask = context.Input<Tensor>("Mask");
    auto* y = context.Output<Tensor>("Out");

    auto* x_data = x->data<T>();
    auto* mask_data = mask->data<T>();
    auto* y_data = y->mutable_data<T>(context.GetPlace());

    auto x_dim = x->dims();
    auto mask_dim = mask->dims();
    auto batches = x_dim[0];
    auto attn_heads = x_dim[1];
    auto query_seq_len = x_dim[2];
    auto key_seq_len = x_dim[3];

    PADDLE_ENFORCE_GT(query_seq_len, 1,
                      platform::errors::InvalidArgument(
                          "Input x's second last dim must be large than 1 but "
                          "received the second last dimension of x is %d",
                          query_seq_len));

    PADDLE_ENFORCE_EQ(key_seq_len >= 32 && key_seq_len < 8192, true,
                      platform::errors::InvalidArgument(
                          "Input x's last dim must be between [32, 8192) "
                          "received the last dimension of x is %d",
                          key_seq_len));

    PADDLE_ENFORCE_EQ(mask_dim[1], 1,
                      platform::errors::InvalidArgument(
                          "Input mask's second dim must be 1 "
                          "received the second dimension of mask is %d",
                          mask_dim[1]));

    // dim of x and mask must be equal
    for (size_t idx = 0; idx < 4; ++idx) {
      if (idx == 1) continue;
      PADDLE_ENFORCE_EQ(
          x_dim[idx], mask_dim[idx],
          platform::errors::InvalidArgument(
              "Input x's %dth dim should be equal with input mask's %dth dim "
              "but "
              "received the %dth dimension of x and mask are not equal "
              "the %dth dim of x is %d, while the %dth dim of mask is %d.",
              idx, idx, idx, idx, x_dim[idx], idx, mask_dim[idx]));
    }

    auto& place = *context.template device_context<Place>().eigen_device();
    auto stream = context.cuda_device_context().stream();

    int pow2_index = get_pow2(key_seq_len);
    const int next_pow2 = 1 << pow2_index;
    int batch_count = batches * attn_heads * query_seq_len;
    int warp_size = (next_pow2 < WARP_SIZE) ? next_pow2 : WARP_SIZE;
    int batches_per_warp = (next_pow2 <= 128) ? 2 : 1;
    // use 128 threads per block to maximum gpu utilization
    constexpr int threads_per_block = 128;

    int warps_per_block = (threads_per_block / warp_size);
    int batches_per_block = warps_per_block * batches_per_warp;
    PADDLE_ENFORCE_EQ(
        query_seq_len % batches_per_block, 0,
        platform::errors::InvalidArgument(
            "The query seq len (third dim of input X) must can divide the "
            "number of batches per block. The query seq len is %d, while "
            "the number of batches per block is %d.",
            query_seq_len, batches_per_block));
    dim3 blocks(query_seq_len / batches_per_block, attn_heads, batches);
    dim3 threads(warp_size, warps_per_block, 1);

    // launch the kernel based on the pow2_index
    switch (pow2_index) {
      case 5:  // 32
        SoftmaxMaskFuseGPUKernel<T, 5><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 6:  // 64
        SoftmaxMaskFuseGPUKernel<T, 6><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 7:  // 128
        SoftmaxMaskFuseGPUKernel<T, 7><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 8:  // 256
        SoftmaxMaskFuseGPUKernel<T, 8><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 9:  // 512
        SoftmaxMaskFuseGPUKernel<T, 9><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 10:  // 1024
        SoftmaxMaskFuseGPUKernel<T, 10><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 11:  // 2048
        SoftmaxMaskFuseGPUKernel<T, 11><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 12:  // 4096
        SoftmaxMaskFuseGPUKernel<T, 12><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      case 13:  // 8192
        SoftmaxMaskFuseGPUKernel<T, 13><<<blocks, threads, 0, stream>>>(
            x_data, mask_data, y_data, batch_count, key_seq_len);
        break;
      default:
        break;
    }
  }
};

template <typename Place, typename T>
class SoftmaxMaskFuseGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* grad_x = context.Output<Tensor>(framework::GradVarName("X"));
    auto* grad_y = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* softmax_rst = context.Input<Tensor>("Softmax");

    auto* grad_x_data = grad_x->mutable_data<T>(context.GetPlace());
    auto* grad_y_data = grad_y->data<T>();
    auto* softmax_rst_data = softmax_rst->data<T>();

    auto y_dim = grad_y->dims();
    auto batches = y_dim[0];
    auto attn_heads = y_dim[1];
    auto query_seq_len = y_dim[2];
    auto key_seq_len = y_dim[3];

    auto& place = *context.template device_context<Place>().eigen_device();
    auto stream = context.cuda_device_context().stream();

    int pow2_index = get_pow2(key_seq_len);
    const int next_pow2 = 1 << pow2_index;
    int batch_count = batches * attn_heads * query_seq_len;
    int warp_size = (next_pow2 < WARP_SIZE) ? next_pow2 : WARP_SIZE;
    int batches_per_warp = (next_pow2 <= 128) ? 2 : 1;
    // use 128 threads per block to maximum gpu utilization
    constexpr int threads_per_block = 128;

    int warps_per_block = (threads_per_block / warp_size);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = batch_count / batches_per_block;
    dim3 threads(warp_size, warps_per_block, 1);

    // launch the kernel based on the pow2_index
    switch (pow2_index) {
      case 5:  // 32
        SoftmaxMaskFuseGradGPUKernel<T, 5><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 6:  // 64
        SoftmaxMaskFuseGradGPUKernel<T, 6><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 7:  // 128
        SoftmaxMaskFuseGradGPUKernel<T, 7><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 8:  // 256
        SoftmaxMaskFuseGradGPUKernel<T, 8><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 9:  // 512
        SoftmaxMaskFuseGradGPUKernel<T, 9><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 10:  // 1024
        SoftmaxMaskFuseGradGPUKernel<T, 10><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 11:  // 2048
        SoftmaxMaskFuseGradGPUKernel<T, 11><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 12:  // 4096
        SoftmaxMaskFuseGradGPUKernel<T, 12><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      case 13:  // 8192
        SoftmaxMaskFuseGradGPUKernel<T, 13><<<blocks, threads, 0, stream>>>(
            grad_y_data, grad_x_data, softmax_rst_data, batch_count,
            key_seq_len);
        break;
      default:
        break;
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    fused_softmax_mask,
    ops::SoftmaxMaskFuseKernel<plat::CUDADeviceContext, plat::float16>,
    ops::SoftmaxMaskFuseKernel<plat::CUDADeviceContext, float>);
REGISTER_OP_CUDA_KERNEL(
    fused_softmax_mask_grad,
    ops::SoftmaxMaskFuseGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::SoftmaxMaskFuseGradKernel<plat::CUDADeviceContext, float>);
