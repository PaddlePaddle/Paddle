#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/fluid/operators/top_k_v2_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

#define FIXED_BLOCK_DIM_BASE(dim, ...) \
  case (dim): {                        \
    constexpr auto kBlockDim = (dim);  \
    __VA_ARGS__;                       \
  } break

#define FIXED_BLOCK_DIM(...)                \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(32, ##__VA_ARGS__)

template <typename DeviceContext, typename T>
class TopkV2OpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");
    auto* indices = ctx.Output<Tensor>("Indices");

    // get the attributes
    int k = static_cast<int>(ctx.Attr<int>("k"));
    int axis = static_cast<int>(ctx.Attr<int>("axis"));
    const bool& sorted = static_cast<bool>(ctx.Attr<bool>("sorted"));
    const bool& largest = static_cast<bool>(ctx.Attr<bool>("largest"));

    // get the input dims
    const auto& in_dims = input->dims();
    // calcluate the real axis
    if (axis < 0) axis += in_dims.size();

    auto* k_t = ctx.Input<Tensor>("K");
    if (k_t) {
      Tensor k_host;
      framework::TensorCopySync(*k_t, platform::CPUPlace(), &k_host);
      k = k_host.data<int>()[0];
      framework::DDim output_dims = output->dims();
      output_dims[axis] = k;
      output->Resize(output_dims);
      indices->Resize(output_dims);
    }

    const auto& out_dims = output->dims();

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    int64_t* indices_data = indices->mutable_data<int64_t>(ctx.GetPlace());

    if (axis == in_dims.size() - 1) {
      // if get the topK from the last axis
      const int64_t& input_height = framework::product(
          framework::slice_ddim(in_dims, 0, in_dims.size() - 1));
      const int64_t& input_width = in_dims[in_dims.size() - 1];
      const auto& dev_ctx = ctx.cuda_device_context();

      if (k > input_width) k = input_width;

      if ((input_width <= 1024 || k >= 128 || k == input_width)) {
        if (SortTopk<T>(dev_ctx, input, input_width, input_height, k, output,
                        indices, largest)) {
          // Successed, return.
          return;
        } else {
          LOG(INFO) << "TopKOP: Some errors happened when use cub sorting, use "
                       "default topk kernel.";
        }
      }

      // NOTE: pass lds and dim same to input width.
      // NOTE: old matrix implementation of stride is different to eigen.
      const int kMaxHeight = 2048;
      int gridx = input_height < kMaxHeight ? input_height : kMaxHeight;
      switch (GetDesiredBlockDim(input_width)) {
        FIXED_BLOCK_DIM(
            KeMatrixTopK<T, 5,
                         kBlockDim><<<gridx, kBlockDim, 0, dev_ctx.stream()>>>(
                output_data, k, indices_data, input_data, input_width,
                input_width, static_cast<int>(k), gridx, input_height,
                largest));
        default:
          PADDLE_THROW(platform::errors::Fatal(
              "the input data shape has error in the topk cuda kernel."));
      }
    } else {
      // if get topK not from the last axis, will tranpose the tensor and get
      // TopK

      // first step, prepare the trans args for the tranpose
      std::vector<int> trans;
      for (int i = 0; i < axis; i++) {
        trans.emplace_back(i);
      }
      trans.emplace_back(in_dims.size() - 1);
      for (int i = axis + 1; i < in_dims.size() - 1; i++) {
        trans.emplace_back(i);
      }
      trans.emplace_back(axis);

      framework::DDim trans_dims(in_dims);
      framework::DDim trans_out_dims(output->dims());
      for (int i = 0; i < trans.size(); i++) {
        trans_dims[i] = in_dims[trans[i]];
        trans_out_dims[i] = out_dims[trans[i]];
      }
      // second step, tranpose the input
      Tensor trans_input;
      trans_input.mutable_data<T>(trans_dims, ctx.GetPlace());
      int ndims = trans.size();
      const auto& dev_ctx = ctx.cuda_device_context();
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, *input,
                                                   &trans_input, trans);
      // third step, calcluate the topk
      // allocate the tmp cuda memory for the tmp result
      Tensor trans_ind;
      trans_ind.mutable_data<int64_t>(trans_out_dims, ctx.GetPlace());
      Tensor trans_out;
      trans_out.mutable_data<T>(trans_out_dims, ctx.GetPlace());

      const int64_t input_height = framework::product(
          framework::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
      const int64_t input_width = trans_dims[trans_dims.size() - 1];

      if (k > input_width) k = input_width;

      if (((input_width <= 1024 && input_height <= 2048) || k >= 128 ||
           k == input_width)) {
        if (SortTopk<T>(dev_ctx, &trans_input, input_width, input_height, k,
                        &trans_out, &trans_ind, largest)) {
          // last step, tranpose back the indices and output
          TransCompute<platform::CUDADeviceContext, int64_t>(
              ndims, dev_ctx, trans_ind, indices, trans);
          TransCompute<platform::CUDADeviceContext, T>(
              ndims, dev_ctx, trans_out, output, trans);
          return;
        } else {
          LOG(INFO) << "TopKOP: Some errors happened when use cub sorting, use "
                       "default topk kernel.";
        }
      }

      const int kMaxHeight = 2048;
      int gridx = input_height < kMaxHeight ? input_height : kMaxHeight;
      switch (GetDesiredBlockDim(input_width)) {
        FIXED_BLOCK_DIM(
            KeMatrixTopK<T, 5,
                         kBlockDim><<<gridx, kBlockDim, 0, dev_ctx.stream()>>>(
                trans_out.data<T>(), k, trans_ind.data<int64_t>(),
                trans_input.data<T>(), input_width, input_width,
                static_cast<int>(k), gridx, input_height, largest));
        default:
          PADDLE_THROW(platform::errors::Fatal(
              "the input data shape has error in the topk cuda kernel."));
      }

      // last step, tranpose back the indices and output
      TransCompute<platform::CUDADeviceContext, int64_t>(
          ndims, dev_ctx, trans_ind, indices, trans);
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, trans_out,
                                                   output, trans);
    }
  }
};

#undef FIXED_BLOCK_DIM_BASE
#undef FIXED_BLOCK_DIM
template <typename DeviceContext, typename T>
class TopkV2OpGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* x = context.Input<Tensor>("X");
    auto* out_grad = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* indices = context.Input<Tensor>("Indices");
    auto* x_grad = context.Output<Tensor>(framework::GradVarName("X"));
    int axis = context.Attr<int>("axis");

    const auto& in_dims = x->dims();
    const auto& out_dims = indices->dims();

    // get the real the axis and the k
    if (axis < 0) axis += in_dims.size();
    const int& k = out_dims[axis];
    const int& raw_height = in_dims[axis];

    // allocate the cuda memory for the x_grad
    T* x_grad_data = x_grad->mutable_data<T>(context.GetPlace());
    const T* out_grad_data = out_grad->data<T>();
    const int64_t* indices_data = indices->data<int64_t>();

    int pre, n, post;
    GetDims(in_dims, axis, &pre, &n, &post);

    // calcluate the block and grid num
    auto& dev_ctx = context.cuda_device_context();
    auto ComputeBlockSize = [](int col) {
      if (col > 512)
        return 1024;
      else if (col > 256 && col <= 512)
        return 512;
      else if (col > 128 && col <= 256)
        return 256;
      else if (col > 64 && col <= 128)
        return 128;
      else
        return 64;
    };
    int block_size = ComputeBlockSize(post * k);
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
    int grid_size = std::min(max_blocks, pre);

    // lanuch the cuda kernel to assign the grad
    AssignGradWithAxis<T><<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
        out_grad_data, indices_data, x_grad_data, pre, post, n, k);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    top_k_v2,
    paddle::operators::TopkV2OpCUDAKernel<paddle::platform::CUDADeviceContext,
                                          float>,
    paddle::operators::TopkV2OpCUDAKernel<paddle::platform::CUDADeviceContext,
                                          double>,
    paddle::operators::TopkV2OpCUDAKernel<paddle::platform::CUDADeviceContext,
                                          int>,
    paddle::operators::TopkV2OpCUDAKernel<paddle::platform::CUDADeviceContext,
                                          int64_t>,
    paddle::operators::TopkV2OpCUDAKernel<paddle::platform::CUDADeviceContext,
                                          paddle::platform::float16>);

REGISTER_OP_CUDA_KERNEL(
    top_k_v2_grad, paddle::operators::TopkV2OpGradCUDAKernel<
                       paddle::platform::CUDADeviceContext, float>,
    paddle::operators::TopkV2OpGradCUDAKernel<
        paddle::platform::CUDADeviceContext, double>,
    paddle::operators::TopkV2OpGradCUDAKernel<
        paddle::platform::CUDADeviceContext, int>,
    paddle::operators::TopkV2OpGradCUDAKernel<
        paddle::platform::CUDADeviceContext, int64_t>,
    paddle::operators::TopkV2OpGradCUDAKernel<
        paddle::platform::CUDADeviceContext, paddle::platform::float16>);
