/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/controlflow/compare_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"

namespace ops = paddle::operators;
namespace plat = paddle::platform;

namespace paddle {
namespace operators {

template <typename Functor, typename InverseFunctor>
class CompareOpKernel<platform::CUDADeviceContext, Functor, InverseFunctor>
    : public framework::OpKernel<typename Functor::ELEM_TYPE> {
 public:
  using InT = typename Functor::ELEM_TYPE;
  using OutT = bool;
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto functor = Functor();
    std::vector<const framework::Tensor*> ins;
    std::vector<framework::Tensor*> outs;
    const auto& cuda_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();

    int axis = PackTensorsIntoVector<OutT>(ctx, &ins, &outs);
    paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary,
                                                   InT, OutT>(
        cuda_ctx, ins, &outs, axis, functor);
  }
};

}  // namespace operators
}  // namespace paddle

#define REGISTER_CUDA_COMPARE_KERNEL(op_type, func)                            \
  REGISTER_OP_CUDA_KERNEL(                                                     \
      op_type,                                                                 \
      ops::CompareOpKernel<plat::CUDADeviceContext, ops::func<bool>, void>,    \
      ops::CompareOpKernel<plat::CUDADeviceContext, ops::func<int16_t>, void>, \
      ops::CompareOpKernel<plat::CUDADeviceContext, ops::func<int>, void>,     \
      ops::CompareOpKernel<plat::CUDADeviceContext, ops::func<int64_t>, void>, \
      ops::CompareOpKernel<plat::CUDADeviceContext, ops::func<float>, void>,   \
      ops::CompareOpKernel<plat::CUDADeviceContext, ops::func<double>, void>);

REGISTER_CUDA_COMPARE_KERNEL(equal, EqualFunctor)
REGISTER_CUDA_COMPARE_KERNEL(not_equal, NotEqualFunctor)
REGISTER_CUDA_COMPARE_KERNEL(less_than, LessThanFunctor)
REGISTER_CUDA_COMPARE_KERNEL(less_equal, LessEqualFunctor)
REGISTER_CUDA_COMPARE_KERNEL(greater_than, GreaterThanFunctor)
REGISTER_CUDA_COMPARE_KERNEL(greater_equal, GreaterEqualFunctor)
#undef REGISTER_CUDA_COMPARE_KERNEL
