/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/controlflow/compare_op.h"

REGISTER_COMPARE_KERNEL(less_than, CUDA, paddle::operators::LessThanFunctor);
REGISTER_COMPARE_KERNEL(less_equal, CUDA, paddle::operators::LessEqualFunctor);
REGISTER_COMPARE_KERNEL(greater_than, CUDA,
                        paddle::operators::GreaterThanFunctor);
REGISTER_COMPARE_KERNEL(greater_equal, CUDA,
                        paddle::operators::GreaterEqualFunctor);
REGISTER_COMPARE_KERNEL(equal, CUDA, paddle::operators::EqualFunctor);
REGISTER_COMPARE_KERNEL(not_equal, CUDA, paddle::operators::NotEqualFunctor);
