#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <vector>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/amp/update_loss_scaling_op.h"
#include "paddle/fluid/platform/enforce.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void GpuUpdateLossScaling(
    const bool* found_inf_data, const T* pre_loss_scaling_data,
    const int* good_in_data, const int* bad_in_data,
    const int incr_every_n_steps, const int decr_every_n_nan_or_inf,
    const float incr_ratio, const float decr_ratio,
    T* updated_loss_scaling_data, int* good_out_data, int* bad_out_data) {
  Update<T>(found_inf_data, pre_loss_scaling_data, good_in_data, bad_in_data,
            incr_every_n_steps, decr_every_n_nan_or_inf, incr_ratio, decr_ratio,
            updated_loss_scaling_data, good_out_data, bad_out_data);
}

template <typename T>
class UpdateLossScalingFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const bool* found_inf_data, const T* pre_loss_scaling_data,
                  const int* good_in_data, const int* bad_in_data,
                  const int incr_every_n_steps,
                  const int decr_every_n_nan_or_inf, const float incr_ratio,
                  const float decr_ratio, T* updated_loss_scaling_data,
                  int* good_out_data, int* bad_out_data) const {
    GpuUpdateLossScaling<T><<<1, 1, 0, dev_ctx.stream()>>>(
        found_inf_data, pre_loss_scaling_data, good_in_data, bad_in_data,
        incr_every_n_steps, decr_every_n_nan_or_inf, incr_ratio, decr_ratio,
        updated_loss_scaling_data, good_out_data, bad_out_data);
  }
};

template <typename T>
class LazyZeroInputs<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const bool* found_inf_data,
                  const std::vector<const framework::Tensor*>& xs,
                  const std::vector<framework::Tensor*>& outs) const {
    const auto gpu_place =
        BOOST_GET_CONST(platform::CUDAPlace, dev_ctx.GetPlace());
    bool has_inf{false};
    memory::Copy(platform::CPUPlace(), &has_inf, gpu_place, found_inf_data,
                 sizeof(bool), dev_ctx.stream());
    dev_ctx.Wait();  // wait async copy
    if (has_inf) {
      VLOG(1) << "-- UpdateLossScaling: Infinite values are found in grads. --";
      for (size_t i = 0; i < xs.size(); ++i) {
        auto* out = outs[i];
        T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());
        int num = out->numel();
        hipMemsetAsync(out_data, 0, num * sizeof(T), dev_ctx.stream());
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using GPU = paddle::platform::CUDADeviceContext;

REGISTER_OP_CUDA_KERNEL(update_loss_scaling,
                        ops::UpdateLossScalingKernel<GPU, float>,
                        ops::UpdateLossScalingKernel<GPU, double>);
