#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <vector>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/amp/update_loss_scaling_op.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void GpuUpdateLossScaling(
    const bool* found_inf_data, const T* pre_loss_scaling_data,
    const int* good_in_data, const int* bad_in_data,
    const int incr_every_n_steps, const int decr_every_n_nan_or_inf,
    const float incr_ratio, const float decr_ratio,
    T* updated_loss_scaling_data, int* good_out_data, int* bad_out_data) {
  Update<T>(found_inf_data, pre_loss_scaling_data, good_in_data, bad_in_data,
            incr_every_n_steps, decr_every_n_nan_or_inf, incr_ratio, decr_ratio,
            updated_loss_scaling_data, good_out_data, bad_out_data);
}

template <typename T>
__global__ void FillIf(T* data, const int64_t num, const T value,
                       const bool* has_inf) {
  if (*has_inf) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < num; i += blockDim.x * gridDim.x) {
      data[i] = value;
    }
  }
}

template <typename T>
class UpdateLossScalingFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const bool* found_inf_data, const T* pre_loss_scaling_data,
                  const int* good_in_data, const int* bad_in_data,
                  const int incr_every_n_steps,
                  const int decr_every_n_nan_or_inf, const float incr_ratio,
                  const float decr_ratio, T* updated_loss_scaling_data,
                  int* good_out_data, int* bad_out_data) const {
    GpuUpdateLossScaling<T><<<1, 1, 0, dev_ctx.stream()>>>(
        found_inf_data, pre_loss_scaling_data, good_in_data, bad_in_data,
        incr_every_n_steps, decr_every_n_nan_or_inf, incr_ratio, decr_ratio,
        updated_loss_scaling_data, good_out_data, bad_out_data);
  }
};

template <typename T>
class LazyZeros<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const bool* found_inf_data,
                  const std::vector<const framework::Tensor*>& xs,
                  const std::vector<framework::Tensor*>& outs) const {
    for (size_t i = 0; i < xs.size(); ++i) {
      auto* out = outs[i];
      T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());
      int64_t num = out->numel();
      int block = 1024;
      int grid = (block - 1 + num) / block;
      FillIf<<<grid, block, 0, dev_ctx.stream()>>>(
          out_data, num, static_cast<T>(0), found_inf_data);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
using GPU = paddle::platform::CUDADeviceContext;

REGISTER_OP_CUDA_KERNEL(update_loss_scaling,
                        ops::UpdateLossScalingKernel<GPU, float>,
                        ops::UpdateLossScalingKernel<GPU, double>,
                        ops::UpdateLossScalingKernel<GPU, plat::float16>);
