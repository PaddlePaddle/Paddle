#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hip/hip_runtime.h>

#include "paddle/fluid/operators/amp/check_finite_and_unscale_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void GpuInverse(const T* s, T* o) {
  *o = Inverse<T>(*s);
}

template <typename T>
__global__ void CheckFiniteAndUnscale(const T* in, const T* scale, int num,
                                      bool* found_inf, T* out) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < num) {
    if (!isfinite(in[idx])) {
      *found_inf = true;
    }
    out[idx] = *found_inf ? in[idx] : in[idx] * (*scale);
  }
}

template <typename T>
class CheckFiniteAndUnscaleGpuKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    const auto xs = ctx.MultiInput<framework::Tensor>("X");
    const auto* scale = ctx.Input<framework::Tensor>("Scale");
    auto outs = ctx.MultiOutput<framework::Tensor>("Out");
    auto* found_inf = ctx.Output<framework::Tensor>("FoundInfinite");

    const T* scale_data = scale->data<T>();
    bool* found_inf_data = found_inf->mutable_data<bool>(dev_ctx.GetPlace());
    hipMemset(found_inf_data, false, found_inf->numel() * sizeof(bool));

    framework::Tensor inverse_scale =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({1}, dev_ctx);
    T* inverse_scale_v = inverse_scale.template data<T>();

    GpuInverse<T><<<1, 1, 0, dev_ctx.stream()>>>(scale_data, inverse_scale_v);

    for (size_t i = 0; i < xs.size(); ++i) {
      const auto* x = xs[i];
      auto* out = outs[i];
      const T* x_data = x->data<T>();
      T* out_data = out->mutable_data<T>(dev_ctx.GetPlace());

      int num = x->numel();
      int block = 1024;
      int grid = (num + block - 1) / block;
      VLOG(3) << "launch kernel";
      CheckFiniteAndUnscale<T><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, inverse_scale_v, num, found_inf_data, out_data);
      VLOG(3) << "finish kernel";
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(check_finite_and_unscale,
                        ops::CheckFiniteAndUnscaleGpuKernel<float>,
                        ops::CheckFiniteAndUnscaleGpuKernel<double>);
