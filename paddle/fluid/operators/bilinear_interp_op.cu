#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/fluid/operators/bilinear_interp_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using framework::Tensor;

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const T ratio_h, const T ratioW) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < nthreads) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;
    int channel_id = out_id_w / out_img_size;

    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int in_img_idy = ratio_h * out_img_idy;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T h1lambda = ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int out_img_idx = tid % out_img_w;
    int in_img_idx = ratioW * out_img_idx;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T w1lambda = ratioW * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                          in_img_idy * in_img_w + in_img_idx];

    // bilinear interpolation
    out[out_id_h * output_w + out_id_w] =
        h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
        h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                    w1lambda * in_pos[h_id * in_img_w + w_id]);
  }
}

template <typename T>
__global__ void KeBilinearInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const T ratio_h, const T ratioW) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < nthreads) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;
    int channel_id = out_id_w / out_img_size;

    int out_img_idy = (out_id_w % out_img_size) / out_img_w;
    int in_img_idy = ratio_h * out_img_idy;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T h1lambda = ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int out_img_idx = tid % out_img_w;
    int in_img_idx = ratioW * out_img_idx;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T w1lambda = ratioW * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    const T* out_pos = &out[out_id_h * output_w + out_id_w];
    atomicAdd(&in_pos[0], h2lambda * w2lambda * out_pos[0]);
    atomicAdd(&in_pos[w_id], h2lambda * w1lambda * out_pos[0]);
    atomicAdd(&in_pos[h_id * in_img_w], h1lambda * w2lambda * out_pos[0]);
    atomicAdd(&in_pos[h_id * in_img_w + w_id],
              h1lambda * w1lambda * out_pos[0]);
  }
}

template <typename T>
class BilinearInterpOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto* input_t = ctx.Input<Tensor>("X");      // float tensor
    auto* output_t = ctx.Output<Tensor>("Out");  // float tensor
    auto* input = input_t->data<T>();

    int out_h = ctx.Attr<int>("out_h");
    int out_w = ctx.Attr<int>("out_w");
    auto out_dims = output_t->dims();
    auto out_size_t = ctx.Input<Tensor>("OutSize");
    if (out_size_t != nullptr) {
      Tensor sizes;
      framework::TensorCopy(*out_size_t, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }
    auto* output = output_t->mutable_data<T>(
        {out_dims[0], out_dims[1], out_h, out_w}, ctx.GetPlace());

    int batch_size = input_t->dims()[0];
    int channels = input_t->dims()[1];
    int in_h = input_t->dims()[2];
    int in_w = input_t->dims()[3];

    int in_hw = in_h * in_w;
    int out_hw = out_h * out_w;
    int in_chw = channels * in_hw;
    int out_chw = channels * out_hw;

    T ratio_h = (out_h > 1) ? static_cast<T>(in_h - 1) / (out_h - 1) : 0.f;
    T ratio_w = (out_w > 1) ? static_cast<T>(in_w - 1) / (out_w - 1) : 0.f;

    if (in_h == out_h && in_w == out_w) {
      memcpy(output, input, input_t->numel() * sizeof(T));
    } else {
      int threadNum = batch_size * out_chw;
      int blocks = (threadNum + 1024 - 1) / 1024;

      KeBilinearInterpFw<
          T><<<blocks, 1024, 0, ctx.cuda_device_context().stream()>>>(
          input, in_h, in_w, batch_size, in_chw, output, out_h, out_w,
          batch_size, out_chw, channels, ratio_h, ratio_w);
    }
  }
};

template <typename T>
class BilinearInterpGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* d_input_t = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* d_output_t = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* d_output = d_output_t->data<T>();
    auto* d_input = d_input_t->mutable_data<T>(ctx.GetPlace());

    auto& device_ctx =
        ctx.template device_context<platform::CUDADeviceContext>();
    math::SetConstant<platform::CUDADeviceContext, T> zero;
    zero(device_ctx, d_input_t, static_cast<T>(0.0));

    int out_h = ctx.Attr<int>("out_h");
    int out_w = ctx.Attr<int>("out_w");

    auto out_size_t = ctx.Input<Tensor>("OutSize");
    if (out_size_t != nullptr) {
      Tensor sizes;
      framework::TensorCopy(*out_size_t, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }

    int batch_size = d_input_t->dims()[0];
    int channels = d_input_t->dims()[1];
    int in_h = d_input_t->dims()[2];
    int in_w = d_input_t->dims()[3];

    int in_hw = in_h * in_w;
    int out_hw = out_h * out_w;
    int in_chw = channels * in_hw;
    int out_chw = channels * out_hw;

    T ratio_h = (out_h > 1) ? static_cast<T>(in_h - 1) / (out_h - 1) : 0.f;
    T ratio_w = (out_w > 1) ? static_cast<T>(in_w - 1) / (out_w - 1) : 0.f;

    if (in_h == out_h && in_w == out_w) {
      memcpy(d_input, d_output, d_input_t->numel() * sizeof(T));
    } else {
      int threadNum = batch_size * out_chw;
      int blocks = (threadNum + 1024 - 1) / 1024;

      KeBilinearInterpBw<
          T><<<blocks, 1024, 0, ctx.cuda_device_context().stream()>>>(
          d_input, in_h, in_w, batch_size, in_chw, d_output, out_h, out_w,
          batch_size, out_chw, channels, ratio_h, ratio_w);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(bilinear_interp,
                        ops::BilinearInterpOpCUDAKernel<float>);
REGISTER_OP_CUDA_KERNEL(bilinear_interp_grad,
                        ops::BilinearInterpGradOpCUDAKernel<float>);
