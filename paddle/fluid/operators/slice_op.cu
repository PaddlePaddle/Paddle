#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/slice_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

template <size_t D>
__global__ void Padding(const paddle::platform::float16* d_out,
                        const int* out_dims, const int* in_dims,
                        const int* offsets, int64_t n,
                        paddle::platform::float16* d_in) {
  int64_t out_idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (out_idx < n) {
    int64_t out_idx_tmp = out_idx;
    int coords[D] = {0};
    for (int i = D - 1; i >= 0; --i) {
      coords[i] = out_idx_tmp % out_dims[i];
      out_idx_tmp /= out_dims[i];
      coords[i] += offsets[i];
    }

    int64_t in_idx = 0;
    for (int i = 0; i < D; ++i) {
      in_idx = in_idx * in_dims[i] + coords[i];
    }

    d_in[in_idx] = d_out[out_idx];
  }
}

template <>
class SliceGradKernel<paddle::platform::CUDADeviceContext,
                      paddle::platform::float16>
    : public framework::OpKernel<paddle::platform::float16> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* d_out = ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* d_in = ctx.Output<framework::Tensor>(framework::GradVarName("Input"));
    d_in->mutable_data<paddle::platform::float16>(ctx.GetPlace());

    auto out_dims = d_out->dims();
    auto in_dims = d_in->dims();
    int rank = out_dims.size();
    std::vector<int> offsets(rank, 0);
    auto axes = ctx.Attr<std::vector<int>>("axes");
    auto starts = ctx.Attr<std::vector<int>>("starts");

    auto list_new_starts_tensor =
        ctx.MultiInput<framework::Tensor>("StartsTensorList");

    if (list_new_starts_tensor.size() > 0) {
      starts = get_new_data_from_tensorlist(list_new_starts_tensor);
    } else if (ctx.HasInput("StartsTensor")) {
      auto* starts_tensor = ctx.Input<framework::Tensor>("StartsTensor");
      starts = get_new_data_from_tensor(starts_tensor);
    }

    for (size_t i = 0; i < starts.size(); ++i) {
      if (starts[i] < 0) {
        starts[i] += in_dims[axes[i]];
      }
      offsets[axes[i]] = std::max(starts[i], 0);
    }

    math::SetConstant<paddle::platform::CUDADeviceContext,
                      paddle::platform::float16>
        set_zero;
    auto& dev_ctx =
        ctx.template device_context<paddle::platform::CUDADeviceContext>();
    set_zero(dev_ctx, d_in, static_cast<paddle::platform::float16>(0));

    int64_t numel = d_out->numel();
    dim3 blocks((numel - 1) / PADDLE_CUDA_NUM_THREADS + 1);
    dim3 threads(PADDLE_CUDA_NUM_THREADS);
    auto stream = ctx.cuda_device_context().stream();

    auto out_shape = framework::vectorize<int>(out_dims);
    thrust::device_vector<int> out_dims_vec(out_shape.begin(), out_shape.end());
    auto in_shape = framework::vectorize<int>(in_dims);
    thrust::device_vector<int> in_dims_vec(in_shape.begin(), in_shape.end());
    thrust::device_vector<int> offsets_vec(offsets.begin(), offsets.end());
    const int* out_dims_ptr = thrust::raw_pointer_cast(out_dims_vec.data());
    const int* in_dims_ptr = thrust::raw_pointer_cast(in_dims_vec.data());
    const int* offsets_ptr = thrust::raw_pointer_cast(offsets_vec.data());

    switch (rank) {
      case 1:
        Padding<1><<<blocks, threads, 0, stream>>>(
            d_out->data<paddle::platform::float16>(), out_dims_ptr, in_dims_ptr,
            offsets_ptr, numel, d_in->data<paddle::platform::float16>());
        break;
      case 2:
        Padding<2><<<blocks, threads, 0, stream>>>(
            d_out->data<paddle::platform::float16>(), out_dims_ptr, in_dims_ptr,
            offsets_ptr, numel, d_in->data<paddle::platform::float16>());
        break;
      case 3:
        Padding<3><<<blocks, threads, 0, stream>>>(
            d_out->data<paddle::platform::float16>(), out_dims_ptr, in_dims_ptr,
            offsets_ptr, numel, d_in->data<paddle::platform::float16>());
        break;
      case 4:
        Padding<4><<<blocks, threads, 0, stream>>>(
            d_out->data<paddle::platform::float16>(), out_dims_ptr, in_dims_ptr,
            offsets_ptr, numel, d_in->data<paddle::platform::float16>());
        break;
      case 5:
        Padding<5><<<blocks, threads, 0, stream>>>(
            d_out->data<paddle::platform::float16>(), out_dims_ptr, in_dims_ptr,
            offsets_ptr, numel, d_in->data<paddle::platform::float16>());
        break;
      case 6:
        Padding<6><<<blocks, threads, 0, stream>>>(
            d_out->data<paddle::platform::float16>(), out_dims_ptr, in_dims_ptr,
            offsets_ptr, numel, d_in->data<paddle::platform::float16>());
        break;
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    slice, ops::SliceKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SliceKernel<paddle::platform::CUDADeviceContext, double>,
    ops::SliceKernel<paddle::platform::CUDADeviceContext, int>,
    ops::SliceKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::SliceKernel<paddle::platform::CUDADeviceContext, plat::float16>);

REGISTER_OP_CUDA_KERNEL(
    slice_grad,
    ops::SliceGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SliceGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::SliceGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::SliceGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::SliceGradKernel<paddle::platform::CUDADeviceContext, plat::float16>);
