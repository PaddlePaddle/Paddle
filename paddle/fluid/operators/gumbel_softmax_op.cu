#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#pragma once

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/gumbel_softmax_op.h"

#if defined(__NVCC__) || defined(__HIPCC__)
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/memory/memcpy.h"

namespace paddle {
namespace operators {

template <typename K, typename V>
using KeyValuePair = hipcub::KeyValuePair<K, V>;

template <typename T>
struct UniformCUDAGenerator {
  T min_, max_;
  unsigned int seed_;
  unsigned int offset_ = 0;
  HOSTDEVICE UniformCUDAGenerator(T min, T max, unsigned int seed)
      : min_(min), max_(max), seed_(seed) {}
  HOSTDEVICE UniformCUDAGenerator(T min, T max, unsigned int seed,
                                  unsigned int offset)
      : min_(min), max_(max), seed_(seed), offset_(offset) {}

  HOSTDEVICE T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n + offset_);
    return dist(rng);
  }
};

template <typename T, size_t BlockDim>
__global__ void OneHotCUDAKernel(const int64_t height, const int64_t width,
                                 const int64_t size_out_axis, const T init,
                                 const T* in, T* out) {
  typedef hipcub::BlockReduce<KeyValuePair<int, T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int64_t idx = blockIdx.x; idx < height; idx += gridDim.x) {
    KeyValuePair<int, T> kv_pair = {-1, init};
    int h = idx / size_out_axis;
    int w = idx % size_out_axis;
    hipcub::ArgMax reducer;
    for (int k = threadIdx.x; k < width; k += blockDim.x) {
      kv_pair = reducer(
          {k, in[h * width * size_out_axis + k * size_out_axis + w]}, kv_pair);
    }
    kv_pair = BlockReduce(temp_storage).Reduce(kv_pair, reducer);
    if (threadIdx.x == 0) {
      int index = static_cast<int>(kv_pair.key);
      out[h * width * size_out_axis + index * size_out_axis + w] = 1;
    }
    __syncthreads();
  }
}

template <typename T>
struct OneHotGenerator<platform::CUDADeviceContext, T> {
  static void Transform(const platform::CUDADeviceContext& context,
                        const Tensor& X, Tensor* Out, int axis) {
    const int size_to_axis = SizeToAxis(axis, X.dims());
    const int size_from_axis = SizeFromAxis(axis, X.dims());
    const int size_out_axis = SizeOutAxis(axis, X.dims());
    constexpr int thread_size = 512;
    int64_t max_grid_dimx = context.GetCUDAMaxGridDimSize().x;
    int64_t height = size_to_axis * size_out_axis;
    int block_size = height < max_grid_dimx ? height : max_grid_dimx;

    Tensor input_tensor;
    input_tensor.mutable_data<T>(Out->dims(), platform::CUDAPlace());
    paddle::framework::TensorCopy(*Out, context.GetPlace(), &input_tensor);
    math::set_constant(context, Out, 0.0);
    OneHotCUDAKernel<
        T, thread_size><<<block_size, thread_size, 0, context.stream()>>>(
        height, size_from_axis / size_out_axis, size_out_axis,
        std::numeric_limits<T>::lowest(), input_tensor.data<T>(),
        Out->data<T>());
  }
};

template <typename T>
__global__ void AddGumbelNoiseCUDAKernel(const T* input_data, T* output_data,
                                         T* noise, const float temperature,
                                         int64_t n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int step = blockDim.x * gridDim.x;
  for (int64_t i = index; i < n; i += step) {
    T gumbel_noise = -log(-log(noise[i]));
    output_data[i] = (gumbel_noise + input_data[i]) / temperature;
  }
}

template <typename T>
struct GumbleNoiseGenerator<platform::CUDADeviceContext, T> {
  static void Transform(const platform::CUDADeviceContext& context,
                        const T* input_data, T* output_data, int size_to_axis,
                        int size_from_axis, const float temperature) {
    Tensor random_tensor;
    int64_t size = size_to_axis * size_from_axis;
    T* random_data =
        random_tensor.mutable_data<T>({size}, platform::CUDAPlace());
    thrust::counting_iterator<int64_t> index_sequence_begin(0);

    // generate gumbel noise
    int device_id =
        BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace()).GetDeviceId();
    auto gen_cuda = framework::GetDefaultCUDAGenerator(device_id);
    if (gen_cuda->GetIsInitPy()) {
      auto seed_offset = gen_cuda->IncrementOffset(1);
      int64_t gen_offset = size * seed_offset.second;
      thrust::transform(
          index_sequence_begin, index_sequence_begin + size,
          thrust::device_ptr<T>(random_data),
          UniformCUDAGenerator<T>(0.00001, 1, seed_offset.first, gen_offset));
    } else {
      const unsigned int seed = std::random_device()();
      thrust::transform(index_sequence_begin, index_sequence_begin + size,
                        thrust::device_ptr<T>(random_data),
                        UniformCUDAGenerator<T>(0.00001, 1, seed));
    }

    // add gumbel noise to X
    const int thread_size = 512;
    int64_t block_size = (size + thread_size) / thread_size;
    AddGumbelNoiseCUDAKernel<
        T><<<block_size, thread_size, 0, context.stream()>>>(
        input_data, output_data, random_data, temperature, size);
  }
};

#endif
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    gumbel_softmax, ops::GumbelSoftmaxKernel<plat::CUDADeviceContext, float>,
    ops::GumbelSoftmaxKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    gumbel_softmax_grad,
    ops::GumbelSoftmaxGradKernel<plat::CUDADeviceContext, float>,
    ops::GumbelSoftmaxGradKernel<plat::CUDADeviceContext, double>);
