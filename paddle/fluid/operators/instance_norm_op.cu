#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/instance_norm_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
static __global__ void repeat_param(const T *input, T *output,
                                    const int repeat_num, const int C) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < repeat_num * C;
       i += blockDim.x * gridDim.x) {
    int index = i % C;
    output[i] = input[index];
  }
}

template <typename T, int BlockDim, bool AVG>
static __global__ void add_param(const T *input, T *output,
                                 const int repeat_num, const int C) {
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ou_storage;
  for (int i = blockIdx.x; i < C; i += gridDim.x) {
    T ou = static_cast<T>(0);
    for (int j = threadIdx.x; j < repeat_num; j += blockDim.x) {
      const int index = j * C + i;
      ou += static_cast<T>(input[index]);
    }
    ou = BlockReduce(ou_storage).Reduce(ou, hipcub::Sum());
    if (threadIdx.x == 0) {
      output[i] = ou;
    }
    __syncthreads();

    if (AVG) {
      output[i] /= repeat_num;
    }
  }
}

template <typename T>
class InstanceNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must be CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));

    auto *x = ctx.Input<Tensor>("X");
    auto &x_dims = x->dims();
    PADDLE_ENFORCE_GE(x_dims.size(), 2,
                      platform::errors::InvalidArgument(
                          "The `shape` in InstanceNormOp is invalid: "
                          "the size of X's dimensions must greater than "
                          "or equal to 2. But received: "
                          "the size of X's dimensions is [%d]",
                          x_dims.size()));
    PADDLE_ENFORCE_LE(x_dims.size(), 5,
                      platform::errors::InvalidArgument(
                          "The `shape` in InstanceNormOp is invalid: "
                          "the size of X's dimensions must smaller than"
                          "or equal to 5. But received: "
                          "the size of X's dimensions is [%d]",
                          x_dims.size()));
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
    int NxC = N * C;
    Tensor x_tmp;
    x_tmp.ShareDataWith(*x).Resize({1, NxC, H, W, D});

    auto *y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t in_param_desc_;

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));

    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    VLOG(3) << "Setting descriptors.";
    std::vector<int> dims;
    std::vector<int> strides;
    dims = {1, NxC, H, W, D};
    strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDeriveBNTensorDescriptor(
            in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    Tensor scale_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    scale_tmp.mutable_data<T>(ctx.GetPlace());
    Tensor bias_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    bias_tmp.mutable_data<T>(ctx.GetPlace());

    const int n = x->numel();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    const int grid = std::min((NxC + block - 1) / block, max_blocks);

    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        scale->data<T>(), scale_tmp.data<T>(), N, C);
    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        bias->data<T>(), bias_tmp.data<T>(), N, C);

    auto handle = dev_ctx.cudnn_handle();

    math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
        functor;

    auto *saved_mean = ctx.Output<Tensor>("SavedMean");
    auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
    saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    functor(dev_ctx, saved_mean, static_cast<BatchNormParamType<T>>(0));
    functor(dev_ctx, saved_variance, static_cast<BatchNormParamType<T>>(0));

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnBatchNormalizationForwardTraining(
            handle, HIPDNN_BATCHNORM_SPATIAL, CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(), data_desc_, x_tmp.template data<T>(),
            data_desc_, y->template mutable_data<T>(ctx.GetPlace()),
            in_param_desc_, scale_tmp.template data<BatchNormParamType<T>>(),
            bias_tmp.template data<BatchNormParamType<T>>(), 0, nullptr,
            nullptr, epsilon,
            saved_mean->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace()),
            saved_variance->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace())));

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
  }
};

template <typename T, int BlockDim>
static __global__ void GradComputeDX(const T *dy,
                                     const BatchNormParamType<T> *scale,
                                     const BatchNormParamType<T> *mean,
                                     const T *x,
                                     const BatchNormParamType<T> *variance,
                                     const int C, const int sample_size,
                                     T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  BatchNormParamType<T> mean_val = mean[ncid];
  BatchNormParamType<T> inv_var_val = variance[ncid];

  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;

  BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
  BatchNormParamType<T> dy_x_sub_mean_sum =
      static_cast<BatchNormParamType<T>>(0);

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    BatchNormParamType<T> dy_i = static_cast<BatchNormParamType<T>>(dy[i]);
    dy_sum += dy_i;
    dy_x_sub_mean_sum +=
        dy_i * (static_cast<BatchNormParamType<T>>(x[i]) - mean_val);
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_x_sub_mean_sum =
      BlockReduce(dy_x_sub_mean_storage).Reduce(dy_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
  }
  __syncthreads();

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    dx[i] =
        (static_cast<BatchNormParamType<T>>(dy[i]) -
         dy_sum_val / static_cast<BatchNormParamType<T>>(sample_size) -
         (static_cast<BatchNormParamType<T>>(x[i]) - mean_val) *
             dy_x_sub_mean_sum_val * inv_var_val * inv_var_val / sample_size) *
        scale[c] * inv_var_val;
  }
}

template <typename T>
class InstanceNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *x = ctx.Input<Tensor>("X");
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    const auto &x_dims = x->dims();

    int N, C, H, W, D;
    ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
    int NxC = N * C;

    Tensor x_tmp, d_y_tmp;
    x_tmp.ShareDataWith(*x).Resize({1, NxC, H, W, D});
    d_y_tmp.ShareDataWith(*d_y).Resize({1, NxC, H, W, D});

    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    d_x->mutable_data<T>(ctx.GetPlace());
    if (d_scale && d_bias) {
      d_scale->mutable_data<T>(ctx.GetPlace());
      d_bias->mutable_data<T>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(
        scale->dims().size(), 1UL,
        platform::errors::InvalidArgument(
            "The `shape` in InstanceNormOp is invalid: "
            "the size of scale's dimensions must be equal to 1. But "
            "received: the size of scale's dimensions"
            "is [%d]",
            scale->dims().size()));
    PADDLE_ENFORCE_EQ(scale->dims()[0], C,
                      platform::errors::InvalidArgument(
                          "The `shape` in InstanceNormOp is invalid: "
                          "the first dimension of scale must be equal to "
                          "Channels([%d]). But received: "
                          "the first dimension of scale is [%d],"
                          "the dimensions of scale is [%s], ",
                          C, scale->dims()[0], scale->dims()));

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    const int n = x->numel();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    const int grid = std::min(NxC, max_blocks);
    const int grid1 = (C + block - 1) / block;

    Tensor scale_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    scale_tmp.mutable_data<T>(ctx.GetPlace());
    Tensor d_scale_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    Tensor d_bias_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        scale->data<T>(), scale_tmp.data<T>(), N, C);

    std::vector<int> dims;
    std::vector<int> strides;
    dims = {1, NxC, H, W, D};
    strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

    if ((H * W * D) == 1) {
      framework::TensorCopy(*d_y, ctx.GetPlace(), d_x);
      math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
          functor;
      functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
      functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
      return;
    }

    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t in_param_desc_;

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));
    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDeriveBNTensorDescriptor(
            in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));

    const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
    const auto *saved_mean_data =
        saved_mean->template data<BatchNormParamType<T>>();
    const auto *saved_var_data =
        saved_var->template data<BatchNormParamType<T>>();
    if (d_scale && d_bias) {
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnBatchNormalizationBackward(
              dev_ctx.cudnn_handle(), HIPDNN_BATCHNORM_SPATIAL,
              CudnnDataType<T>::kOne(), CudnnDataType<T>::kZero(),
              CudnnDataType<T>::kOne(), CudnnDataType<T>::kZero(), data_desc_,
              x_tmp.template data<T>(), data_desc_, d_y_tmp.template data<T>(),
              data_desc_, d_x->template mutable_data<T>(ctx.GetPlace()),
              in_param_desc_, scale_tmp.template data<BatchNormParamType<T>>(),
              d_scale_tmp.template mutable_data<BatchNormParamType<T>>(
                  ctx.GetPlace()),
              d_bias_tmp.template mutable_data<BatchNormParamType<T>>(
                  ctx.GetPlace()),
              epsilon, saved_mean_data, saved_var_data));
    } else {
      if (d_x) {
        GradComputeDX<T, block><<<NxC, block, 0, dev_ctx.stream()>>>(
            d_y->data<T>(), scale->data<BatchNormParamType<T>>(),
            saved_mean_data, x->data<T>(), saved_var_data, C, H * W * D,
            d_x->data<T>());
      }
    }

    if (d_scale && d_bias) {
      add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
          d_scale_tmp.data<T>(), d_scale->data<T>(), N, C);
      add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
          d_bias_tmp.data<T>(), d_bias->data<T>(), N, C);
    }

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
  }
};

static __device__ __forceinline__ float real_sqrt(float x) {
  return 1. / sqrtf(x);
}
static __device__ __forceinline__ double real_sqrt(double x) {
  return 1. / sqrt(x);
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDX(const T *x, const T *mean,
                                    const T *variance, const T *ddx,
                                    const T *dy, const T *scale,
                                    const T *ddscale, int C, int sample_size,
                                    const double epsilon, T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ T dy_sum_val;
  __shared__ T ddx_sum_val;
  __shared__ T dy_mul_ddx_sum_val;
  __shared__ T dy_mul_x_sub_mean_sum_val;
  __shared__ T ddx_mul_x_sub_mean_sum_val;

  T dy_sum = 0;
  T ddx_sum = 0;
  T dy_mul_ddx_sum = 0;
  T dy_mul_x_sub_mean_sum = 0;
  T ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T ddx_i = ddx[i];
    T dy_i = dy[i];
    T tmp = x[i] - mean_val;

    dy_sum += dy_i;
    ddx_sum += ddx_i;
    dy_mul_ddx_sum += (ddx_i * dy_i);

    dy_mul_x_sub_mean_sum += (dy_i * tmp);
    ddx_mul_x_sub_mean_sum += (ddx_i * tmp);
  }

  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  dy_mul_ddx_sum =
      BlockReduce(dy_mul_ddx_storage).Reduce(dy_mul_ddx_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    ddx_sum_val = ddx_sum;
    dy_mul_ddx_sum_val = dy_mul_ddx_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dx[i] +=
          ((x[i] - mean_val) * var_val * var_val * var_val / sample_size *
               (ddx_sum_val * dy_sum_val / sample_size - dy_mul_ddx_sum_val +
                3. * dy_mul_x_sub_mean_sum_val * var_val *
                    ddx_mul_x_sub_mean_sum_val * var_val / sample_size) +
           ddx_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
               var_val * (dy_sum_val / sample_size - dy[i]) +
           dy_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
               var_val * (ddx_sum_val / sample_size - ddx[i])) *
          scale[c];
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dx[i] += (dy[i] * var_val - dy_sum_val / sample_size * var_val -
                (x[i] - mean_val) * var_val * dy_mul_x_sub_mean_sum_val *
                    var_val / sample_size) *
               ddscale[c];
    }
  }
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDDY(const T *x, const T *mean,
                                     const T *variance, const T *ddscale,
                                     const T *ddbias, const T *ddx,
                                     const T *scale, int C, int sample_size,
                                     const double epsilon, T *ddy) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ T ddx_sum_val;
  __shared__ T ddx_mul_x_sub_mean_sum_val;

  T ddx_sum = 0;
  T ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T ddx_i = ddx[i];
    ddx_sum += ddx_i;
    ddx_mul_x_sub_mean_sum += (ddx_i * (x[i] - mean_val));
  }
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    ddx_sum_val = ddx_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += scale[c] * var_val *
                (ddx[i] - ddx_sum_val / sample_size -
                 (x[i] - mean_val) * var_val * ddx_mul_x_sub_mean_sum_val *
                     var_val / sample_size);
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += (x[i] - mean_val) * var_val * ddscale[c];
    }
  }
  __syncthreads();
  if (ddbias != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += ddbias[c];
    }
  }
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDScale(const T *x, const T *mean,
                                        const T *variance, const T *ddx,
                                        const T *dy, int C, int sample_size,
                                        const double epsilon, T *dscale) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage dscale_tmp_storage;
  __shared__ T dy_sum_val;
  __shared__ T dy_mul_x_sub_mean_sum_val;

  T dy_sum = 0;
  T dy_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T dy_i = dy[i];
    dy_sum += dy_i;
    dy_mul_x_sub_mean_sum += (dy_i * (x[i] - mean_val));
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    T dscale_tmp = 0;
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dscale_tmp +=
          ddx[i] * var_val * (dy[i] - dy_sum_val / sample_size -
                              dy_mul_x_sub_mean_sum_val * (x[i] - mean_val) *
                                  var_val * var_val / sample_size);
    }
    dscale_tmp = BlockReduce(dscale_tmp_storage).Reduce(dscale_tmp, hipcub::Sum());

    if (threadIdx.x == 0) {
      dscale[ncid] += dscale_tmp;
    }
    __syncthreads();
  }
}

template <typename T>
class InstanceNormDoubleGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto *X = ctx.Input<Tensor>("X");
    const auto *Scale = ctx.Input<Tensor>("Scale");
    const auto *dY = ctx.Input<Tensor>("DY");
    const auto *Saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *Saved_variance = ctx.Input<Tensor>("SavedVariance");
    const auto *running_mean = ctx.Input<Tensor>("Mean");
    const auto *running_var = ctx.Input<Tensor>("Variance");
    const auto *ddX = ctx.Input<Tensor>("DDX");
    const auto *ddScale = ctx.Input<Tensor>("DDScale");
    const auto *ddBias = ctx.Input<Tensor>("DDBias");
    const double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));

    auto *dX = ctx.Output<Tensor>("DX");
    auto *dScale = ctx.Output<Tensor>("DScale");
    auto *ddY = ctx.Output<Tensor>("DDY");

    const T *x_data = X->data<T>();
    const T *scale_data = Scale->data<T>();
    const T *dy_data = dY->data<T>();
    const T *ddx_data = (ddX == nullptr ? nullptr : ddX->data<T>());

    const T *ddscale_data = (ddScale == nullptr ? nullptr : ddScale->data<T>());
    const T *ddbias_data = (ddScale == nullptr ? nullptr : ddBias->data<T>());

    const T *mean_data = Saved_mean->data<T>();
    const T *variance_data = Saved_variance->data<T>();

    auto &x_dims = X->dims();
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
    int NxC = N * C;
    const int n = X->numel();
    int sample_size = n / N / C;

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    const int grid = NxC;
    const int grid1 = (C + block - 1) / block;

    math::SetConstant<platform::CUDADeviceContext, T> set_zero;

    if (dX) {
      T *dx_data = dX->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, dX, static_cast<T>(0));
      DoubleGradComputeDX<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, mean_data, variance_data, ddx_data, dy_data, scale_data,
          ddscale_data, C, sample_size, epsilon, dx_data);
    }
    if (dScale) {
      Tensor dscale_tmp =
          ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
      set_zero(dev_ctx, &dscale_tmp, static_cast<T>(0));
      T *dscale_tmp_data = dscale_tmp.mutable_data<T>(ctx.GetPlace());

      T *dscale_data = dScale->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, dScale, static_cast<T>(0));
      DoubleGradComputeDScale<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, mean_data, variance_data, ddx_data, dy_data, C, sample_size,
          epsilon, dscale_tmp_data);
      add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
          dscale_tmp.data<T>(), dScale->data<T>(), N, C);
    }
    if (ddY) {
      T *ddy_data = ddY->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, ddY, static_cast<T>(0));
      DoubleGradComputeDDY<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, mean_data, variance_data, ddscale_data, ddbias_data, ddx_data,
          scale_data, C, sample_size, epsilon, ddy_data);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    instance_norm, ops::InstanceNormKernel<plat::CUDADeviceContext, float>,
    ops::InstanceNormKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    instance_norm_grad,
    ops::InstanceNormGradKernel<plat::CUDADeviceContext, float>,
    ops::InstanceNormGradKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    instance_norm_grad_grad,
    ops::InstanceNormDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                      float>,
    ops::InstanceNormDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                      double>);
