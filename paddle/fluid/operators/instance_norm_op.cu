#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/instance_norm_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
static __global__ void repeat_param(const T *input, T *output,
                                    const int repeat_num, const int C) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < repeat_num * C;
       i += blockDim.x * gridDim.x) {
    int index = i % C;
    output[i] = input[index];
  }
}

template <typename T, int BlockDim>
static __global__ void add_param(const T *input, T *output,
                                 const int repeat_num, const int C) {
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ou_storage;
  for (int i = blockIdx.x; i < C; i += gridDim.x) {
    T ou = static_cast<T>(0);
    for (int j = threadIdx.x; j < repeat_num; j += blockDim.x) {
      const int index = j * C + i;
      ou += static_cast<T>(input[index]);
    }
    ou = BlockReduce(ou_storage).Reduce(ou, hipcub::Sum());
    if (threadIdx.x == 0) {
      output[i] = ou;
    }
    __syncthreads();
  }
}

template <typename T>
class InstanceNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must be CUDAPlace.");
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");

    auto *x = ctx.Input<Tensor>("X");
    auto &x_dims = x->dims();
    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
    int NxC = N * C;
    Tensor x_tmp;
    x_tmp.ShareDataWith(*x).Resize({1, NxC, H, W, D});

    auto *y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t in_param_desc_;

    CUDNN_ENFORCE(platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_ENFORCE(
        platform::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));

    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    VLOG(3) << "Setting descriptors.";
    std::vector<int> dims;
    std::vector<int> strides;
    dims = {1, NxC, H, W, D};
    strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
    CUDNN_ENFORCE(platform::dynload::hipdnnDeriveBNTensorDescriptor(
        in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    Tensor scale_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    scale_tmp.mutable_data<T>(ctx.GetPlace());
    Tensor bias_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    bias_tmp.mutable_data<T>(ctx.GetPlace());

    const int n = x->numel();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    const int grid = std::min(NxC, max_blocks);

    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        scale->data<T>(), scale_tmp.data<T>(), N, C);
    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        bias->data<T>(), bias_tmp.data<T>(), N, C);

    auto handle = dev_ctx.cudnn_handle();
    math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
        functor;

    if (is_test || use_global_stats) {
      const auto *est_mean = ctx.Input<Tensor>("Mean");
      const auto *est_var = ctx.Input<Tensor>("Variance");

      PADDLE_ENFORCE_EQ(est_mean->dims().size(), 1UL);
      PADDLE_ENFORCE_EQ(est_var->dims().size(), 1UL);
      PADDLE_ENFORCE_EQ(est_mean->dims()[0], NxC);
      PADDLE_ENFORCE_EQ(est_var->dims()[0], NxC);

      CUDNN_ENFORCE(platform::dynload::hipdnnBatchNormalizationForwardInference(
          handle, HIPDNN_BATCHNORM_SPATIAL, CudnnDataType<T>::kOne(),
          CudnnDataType<T>::kZero(), data_desc_, x_tmp.template data<T>(),
          data_desc_, y->template mutable_data<T>(ctx.GetPlace()),
          in_param_desc_, scale_tmp.template data<BatchNormParamType<T>>(),
          bias_tmp.template data<BatchNormParamType<T>>(),
          est_mean->template data<BatchNormParamType<T>>(),
          est_var->template data<BatchNormParamType<T>>(), epsilon));
    } else {
      auto *mean_out = ctx.Output<Tensor>("MeanOut");
      auto *variance_out = ctx.Output<Tensor>("VarianceOut");
      mean_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      variance_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

      auto *saved_mean = ctx.Output<Tensor>("SavedMean");
      auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
      saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      functor(dev_ctx, saved_mean, static_cast<BatchNormParamType<T>>(0));
      functor(dev_ctx, saved_variance, static_cast<BatchNormParamType<T>>(0));

      double factor = 1. - momentum;
      CUDNN_ENFORCE(platform::dynload::hipdnnBatchNormalizationForwardTraining(
          handle, HIPDNN_BATCHNORM_SPATIAL, CudnnDataType<T>::kOne(),
          CudnnDataType<T>::kZero(), data_desc_, x_tmp.template data<T>(),
          data_desc_, y->template mutable_data<T>(ctx.GetPlace()),
          in_param_desc_, scale_tmp.template data<BatchNormParamType<T>>(),
          bias_tmp.template data<BatchNormParamType<T>>(), factor,
          mean_out->template mutable_data<BatchNormParamType<T>>(
              ctx.GetPlace()),
          variance_out->template mutable_data<BatchNormParamType<T>>(
              ctx.GetPlace()),
          epsilon, saved_mean->template mutable_data<BatchNormParamType<T>>(
                       ctx.GetPlace()),
          saved_variance->template mutable_data<BatchNormParamType<T>>(
              ctx.GetPlace())));
    }

    CUDNN_ENFORCE(platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_ENFORCE(
        platform::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
  }
};

template <typename T>
static __global__ void INBwdData(const T *dy,
                                 const BatchNormParamType<T> *scale,
                                 const BatchNormParamType<T> *variance,
                                 const double epsilon, const int NxC,
                                 const int C, const int HxW, const int num,
                                 T *dx) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    const int nc = i / HxW % NxC;
    const int c = nc % C;
    BatchNormParamType<T> inv_var = 1.0 / sqrt(variance[nc] + epsilon);
    dx[i] = static_cast<T>(static_cast<BatchNormParamType<T>>(dy[i]) *
                           scale[c] * inv_var);
  }
}

template <typename T, int BlockDim>
static __global__ void INBwdScaleBias(const T *dy, const T *x,
                                      const BatchNormParamType<T> *mean,
                                      const BatchNormParamType<T> *variance,
                                      const double epsilon, const int N,
                                      const int C, const int HxW,
                                      BatchNormParamType<T> *d_scale,
                                      BatchNormParamType<T> *d_bias) {
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ds_storage;
  __shared__ typename BlockReduce::TempStorage db_storage;

  const int inner_size = N * HxW;
  for (int i = blockIdx.x; i < C; i += gridDim.x) {
    BatchNormParamType<T> ds_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> db_sum = static_cast<BatchNormParamType<T>>(0);

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int stats_index = j / HxW * C + i;
      const int index = stats_index * HxW + j % HxW;

      BatchNormParamType<T> var_i = 1.0 / sqrt(variance[stats_index] + epsilon);
      ds_sum +=
          static_cast<BatchNormParamType<T>>(dy[index]) *
          static_cast<BatchNormParamType<T>>(x[index] - mean[stats_index]) *
          static_cast<BatchNormParamType<T>>(var_i);
      db_sum += static_cast<BatchNormParamType<T>>(dy[index]);
    }
    ds_sum = BlockReduce(ds_storage).Reduce(ds_sum, hipcub::Sum());
    db_sum = BlockReduce(db_storage).Reduce(db_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      d_scale[i] = ds_sum;
      d_bias[i] = db_sum;
    }
    __syncthreads();
  }
}

template <typename T>
class InstanceNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *saved_variance = ctx.Input<Tensor>("SavedVariance");
    const auto *x = ctx.Input<Tensor>("X");
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));

    const auto &x_dims = x->dims();

    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
    int NxC = N * C;

    Tensor x_tmp, d_y_tmp;
    x_tmp.ShareDataWith(*x).Resize({1, NxC, H, W, D});
    d_y_tmp.ShareDataWith(*d_y).Resize({1, NxC, H, W, D});

    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    d_x->mutable_data<T>(ctx.GetPlace());
    if (d_scale && d_bias) {
      d_scale->mutable_data<T>(ctx.GetPlace());
      d_bias->mutable_data<T>(ctx.GetPlace());
    }
    PADDLE_ENFORCE(scale->dims().size(), 1UL);
    PADDLE_ENFORCE(scale->dims()[0], C);

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    const int n = x->numel();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    const int grid = std::min(NxC, max_blocks);
    const int grid1 = (n + block - 1) / block;

    Tensor scale_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    scale_tmp.mutable_data<T>(ctx.GetPlace());
    Tensor d_scale_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    Tensor d_bias_tmp =
        ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        scale->data<T>(), scale_tmp.data<T>(), N, C);

    std::vector<int> dims;
    std::vector<int> strides;
    dims = {1, NxC, H, W, D};
    strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

    if (!use_global_stats) {
      if ((H * W * D) == 1) {
        framework::TensorCopy(*d_y, ctx.GetPlace(), d_x);
        math::SetConstant<platform::CUDADeviceContext, BatchNormParamType<T>>
            functor;
        functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
        functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
        return;
      }

      hipdnnTensorDescriptor_t data_desc_;
      hipdnnTensorDescriptor_t in_param_desc_;

      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));
      if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
        LOG(ERROR) << "Provided epsilon is smaller than "
                   << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                   << "HIPDNN_BN_MIN_EPSILON instead.";
      }
      epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

      CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_, CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4, dims.data(), strides.data()));
      CUDNN_ENFORCE(platform::dynload::hipdnnDeriveBNTensorDescriptor(
          in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));

      const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
      const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
      const void *saved_mean_data =
          saved_mean->template data<BatchNormParamType<T>>();
      const void *saved_var_data =
          saved_var->template data<BatchNormParamType<T>>();
      CUDNN_ENFORCE(platform::dynload::hipdnnBatchNormalizationBackward(
          dev_ctx.cudnn_handle(), HIPDNN_BATCHNORM_SPATIAL,
          CudnnDataType<T>::kOne(), CudnnDataType<T>::kZero(),
          CudnnDataType<T>::kOne(), CudnnDataType<T>::kZero(), data_desc_,
          x_tmp.template data<T>(), data_desc_, d_y_tmp.template data<T>(),
          data_desc_, d_x->template mutable_data<T>(ctx.GetPlace()),
          in_param_desc_, scale_tmp.template data<BatchNormParamType<T>>(),
          d_scale_tmp.template mutable_data<BatchNormParamType<T>>(
              ctx.GetPlace()),
          d_bias_tmp.template mutable_data<BatchNormParamType<T>>(
              ctx.GetPlace()),
          epsilon, saved_mean_data, saved_var_data));

      add_param<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          d_scale_tmp.data<T>(), d_scale->data<T>(), N, C);
      add_param<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          d_bias_tmp.data<T>(), d_bias->data<T>(), N, C);
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
    } else {
      const auto *running_mean = ctx.Input<Tensor>("Mean");
      const auto *running_variance = ctx.Input<Tensor>("Variance");

      const auto *running_mean_data =
          running_mean->template data<BatchNormParamType<T>>();
      const auto *running_var_data =
          running_variance->template data<BatchNormParamType<T>>();
      if (d_x) {
        INBwdData<T><<<grid1, block, 0, dev_ctx.stream()>>>(
            d_y_tmp.data<T>(), scale->data<BatchNormParamType<T>>(),
            running_var_data, epsilon, NxC, C, H * W, n, d_x->data<T>());
      }
      if (d_scale && d_bias) {
        INBwdScaleBias<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
            d_y_tmp.data<T>(), x_tmp.data<T>(), running_mean_data,
            running_var_data, epsilon, N, C, H * W,
            d_scale->data<BatchNormParamType<T>>(),
            d_bias->data<BatchNormParamType<T>>());
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    instance_norm, ops::InstanceNormKernel<plat::CUDADeviceContext, float>,
    ops::InstanceNormKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    instance_norm_grad,
    ops::InstanceNormGradKernel<plat::CUDADeviceContext, float>,
    ops::InstanceNormGradKernel<plat::CUDADeviceContext, double>);
// REGISTER_OP_CUDA_KERNEL(
//    instance_norm_grad_grad,
//    ops::InstanceNormDoubleGradKernel<paddle::platform::CUDADeviceContext,
//    float>,
//    ops::InstanceNormDoubleGradKernel<paddle::platform::CUDADeviceContext,
//    double>);
