#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/instance_norm_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
static __global__ void repeat_param(const T *input, T *output,
                                    const int repeat_num, const int C) {
  CUDA_KERNEL_LOOP(i, repeat_num * C) {
    int index = i % C;
    output[i] = input[index];
  }
}

template <typename T, int BlockDim, bool AVG>
static __global__ void add_param(const T *input, T *output,
                                 const int repeat_num, const int C) {
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ou_storage;
  for (int i = blockIdx.x; i < C; i += gridDim.x) {
    T ou = static_cast<T>(0);
    for (int j = threadIdx.x; j < repeat_num; j += blockDim.x) {
      const int index = j * C + i;
      ou += static_cast<T>(input[index]);
    }
    ou = BlockReduce(ou_storage).Reduce(ou, hipcub::Sum());
    if (threadIdx.x == 0) {
      output[i] = ou;
    }
    __syncthreads();

    if (AVG) {
      output[i] /= repeat_num;
    }
  }
}

template <typename T, int BlockDim>
static __global__ void GradComputeDX(const T *dy,
                                     const BatchNormParamType<T> *scale,
                                     const BatchNormParamType<T> *mean,
                                     const T *x,
                                     const BatchNormParamType<T> *variance,
                                     const int C, const int sample_size,
                                     T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  BatchNormParamType<T> mean_val = mean[ncid];
  BatchNormParamType<T> inv_var_val = variance[ncid];

  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;

  BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
  BatchNormParamType<T> dy_x_sub_mean_sum =
      static_cast<BatchNormParamType<T>>(0);

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    BatchNormParamType<T> dy_i = static_cast<BatchNormParamType<T>>(dy[i]);
    dy_sum += dy_i;
    dy_x_sub_mean_sum +=
        dy_i * (static_cast<BatchNormParamType<T>>(x[i]) - mean_val);
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_x_sub_mean_sum =
      BlockReduce(dy_x_sub_mean_storage).Reduce(dy_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
  }
  __syncthreads();

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    dx[i] =
        (static_cast<BatchNormParamType<T>>(dy[i]) -
         dy_sum_val / static_cast<BatchNormParamType<T>>(sample_size) -
         (static_cast<BatchNormParamType<T>>(x[i]) - mean_val) *
             dy_x_sub_mean_sum_val * inv_var_val * inv_var_val / sample_size) *
        scale[c] * inv_var_val;
  }
}

static __device__ __forceinline__ float real_sqrt(float x) {
  return 1. / sqrtf(x);
}
static __device__ __forceinline__ double real_sqrt(double x) {
  return 1. / sqrt(x);
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDX(const T *x, const T *mean,
                                    const T *variance, const T *ddx,
                                    const T *dy, const T *scale,
                                    const T *ddscale, int C, int sample_size,
                                    const double epsilon, T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ T dy_sum_val;
  __shared__ T ddx_sum_val;
  __shared__ T dy_mul_ddx_sum_val;
  __shared__ T dy_mul_x_sub_mean_sum_val;
  __shared__ T ddx_mul_x_sub_mean_sum_val;

  T dy_sum = 0;
  T ddx_sum = 0;
  T dy_mul_ddx_sum = 0;
  T dy_mul_x_sub_mean_sum = 0;
  T ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T ddx_i = ddx[i];
    T dy_i = dy[i];
    T tmp = x[i] - mean_val;

    dy_sum += dy_i;
    ddx_sum += ddx_i;
    dy_mul_ddx_sum += (ddx_i * dy_i);

    dy_mul_x_sub_mean_sum += (dy_i * tmp);
    ddx_mul_x_sub_mean_sum += (ddx_i * tmp);
  }

  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  dy_mul_ddx_sum =
      BlockReduce(dy_mul_ddx_storage).Reduce(dy_mul_ddx_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    ddx_sum_val = ddx_sum;
    dy_mul_ddx_sum_val = dy_mul_ddx_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dx[i] +=
          ((x[i] - mean_val) * var_val * var_val * var_val / sample_size *
               (ddx_sum_val * dy_sum_val / sample_size - dy_mul_ddx_sum_val +
                3. * dy_mul_x_sub_mean_sum_val * var_val *
                    ddx_mul_x_sub_mean_sum_val * var_val / sample_size) +
           ddx_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
               var_val * (dy_sum_val / sample_size - dy[i]) +
           dy_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
               var_val * (ddx_sum_val / sample_size - ddx[i])) *
          scale[c];
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dx[i] += (dy[i] * var_val - dy_sum_val / sample_size * var_val -
                (x[i] - mean_val) * var_val * dy_mul_x_sub_mean_sum_val *
                    var_val / sample_size) *
               ddscale[c];
    }
  }
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDDY(const T *x, const T *mean,
                                     const T *variance, const T *ddscale,
                                     const T *ddbias, const T *ddx,
                                     const T *scale, int C, int sample_size,
                                     const double epsilon, T *ddy) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ T ddx_sum_val;
  __shared__ T ddx_mul_x_sub_mean_sum_val;

  T ddx_sum = 0;
  T ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T ddx_i = ddx[i];
    ddx_sum += ddx_i;
    ddx_mul_x_sub_mean_sum += (ddx_i * (x[i] - mean_val));
  }
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    ddx_sum_val = ddx_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += scale[c] * var_val *
                (ddx[i] - ddx_sum_val / sample_size -
                 (x[i] - mean_val) * var_val * ddx_mul_x_sub_mean_sum_val *
                     var_val / sample_size);
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += (x[i] - mean_val) * var_val * ddscale[c];
    }
  }
  __syncthreads();
  if (ddbias != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += ddbias[c];
    }
  }
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDScale(const T *x, const T *mean,
                                        const T *variance, const T *ddx,
                                        const T *dy, int C, int sample_size,
                                        const double epsilon, T *dscale) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage dscale_tmp_storage;
  __shared__ T dy_sum_val;
  __shared__ T dy_mul_x_sub_mean_sum_val;

  T dy_sum = 0;
  T dy_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T dy_i = dy[i];
    dy_sum += dy_i;
    dy_mul_x_sub_mean_sum += (dy_i * (x[i] - mean_val));
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    T dscale_tmp = 0;
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dscale_tmp +=
          ddx[i] * var_val * (dy[i] - dy_sum_val / sample_size -
                              dy_mul_x_sub_mean_sum_val * (x[i] - mean_val) *
                                  var_val * var_val / sample_size);
    }
    dscale_tmp = BlockReduce(dscale_tmp_storage).Reduce(dscale_tmp, hipcub::Sum());

    if (threadIdx.x == 0) {
      dscale[ncid] += dscale_tmp;
    }
    __syncthreads();
  }
}

template <typename T>
class InstanceNormDoubleGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto *X = ctx.Input<Tensor>("X");
    const auto *Scale = ctx.Input<Tensor>("Scale");
    const auto *dY = ctx.Input<Tensor>("DY");
    const auto *Saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *Saved_variance = ctx.Input<Tensor>("SavedVariance");
    const auto *running_mean = ctx.Input<Tensor>("Mean");
    const auto *running_var = ctx.Input<Tensor>("Variance");
    const auto *ddX = ctx.Input<Tensor>("DDX");
    const auto *ddScale = ctx.Input<Tensor>("DDScale");
    const auto *ddBias = ctx.Input<Tensor>("DDBias");
    const double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));

    auto *dX = ctx.Output<Tensor>("DX");
    auto *dScale = ctx.Output<Tensor>("DScale");
    auto *ddY = ctx.Output<Tensor>("DDY");

    const T *x_data = X->data<T>();
    const T *dy_data = dY->data<T>();
    const T *ddx_data = (ddX == nullptr ? nullptr : ddX->data<T>());

    const T *ddscale_data = (ddScale == nullptr ? nullptr : ddScale->data<T>());
    const T *ddbias_data = (ddScale == nullptr ? nullptr : ddBias->data<T>());

    const T *mean_data = Saved_mean->data<T>();
    const T *variance_data = Saved_variance->data<T>();

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    phi::funcs::SetConstant<platform::CUDADeviceContext, T> set_zero;

    auto &x_dims = X->dims();
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
    int NxC = N * C;
    const int n = X->numel();
    int sample_size = n / N / C;

    Tensor scale_tmp;
    if (!Scale) {
      scale_tmp.mutable_data<T>({C}, ctx.GetPlace());
      set_zero(dev_ctx, &scale_tmp, static_cast<T>(1));
    }
    const T *scale_data = Scale ? Scale->data<T>() : scale_tmp.data<T>();

    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(max_threads / block, 1);
    const int grid = NxC;
    const int grid1 = (C + block - 1) / block;

    if (dX) {
      T *dx_data = dX->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, dX, static_cast<T>(0));
      DoubleGradComputeDX<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, mean_data, variance_data, ddx_data, dy_data, scale_data,
          ddscale_data, C, sample_size, epsilon, dx_data);
    }
    if (dScale) {
      Tensor dscale_tmp =
          ctx.AllocateTmpTensor<T, platform::CUDADeviceContext>({NxC}, dev_ctx);
      set_zero(dev_ctx, &dscale_tmp, static_cast<T>(0));
      T *dscale_tmp_data = dscale_tmp.mutable_data<T>(ctx.GetPlace());

      T *dscale_data = dScale->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, dScale, static_cast<T>(0));
      DoubleGradComputeDScale<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, mean_data, variance_data, ddx_data, dy_data, C, sample_size,
          epsilon, dscale_tmp_data);
      add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
          dscale_tmp.data<T>(), dScale->data<T>(), N, C);
    }
    if (ddY) {
      T *ddy_data = ddY->mutable_data<T>(ctx.GetPlace());
      set_zero(dev_ctx, ddY, static_cast<T>(0));
      DoubleGradComputeDDY<T, block><<<grid, block, 0, dev_ctx.stream()>>>(
          x_data, mean_data, variance_data, ddscale_data, ddbias_data, ddx_data,
          scale_data, C, sample_size, epsilon, ddy_data);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(instance_norm_grad_grad,
                        ops::InstanceNormDoubleGradKernel<
                            paddle::platform::CUDADeviceContext, float>);
#else
REGISTER_OP_CUDA_KERNEL(
    instance_norm_grad_grad,
    ops::InstanceNormDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                      float>,
    ops::InstanceNormDoubleGradKernel<paddle::platform::CUDADeviceContext,
                                      double>);
#endif
