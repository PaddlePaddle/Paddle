#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include <unsupported/Eigen/SpecialFunctions>
#include "paddle/fluid/operators/lgamma_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct LgammaGradFunctorCUDA {
  LgammaGradFunctorCUDA(const T* dout, const T* x, T* output, int64_t numel)
      : dout_(dout), x_(x), output_(output), numel_(numel) {}

  HOSTDEVICE void operator()(int64_t idx) const {
    output_[idx] = dout_[idx] / Eigen::numext::digamma(x_[idx]);
  }

 private:
  const T* dout_;
  const T* x_;
  T* output_;
  int64_t numel_;
};

template <typename T>
class LgammaGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    const framework::Tensor* d_out =
        ctx.Input<framework::Tensor>(framework::GradVarName("Out"));
    const framework::Tensor* x = ctx.Input<framework::Tensor>("X");
    framework::Tensor* d_x =
        ctx.Output<framework::Tensor>(framework::GradVarName("X"));

    auto numel = d_out->numel();
    auto* dout_data = d_out->data<T>();
    auto* x_data = x->data<T>();
    auto* dx_data = d_x->mutable_data<T>(
        ctx.GetPlace(), static_cast<size_t>(numel * sizeof(T)));

    auto& dev_ctx = ctx.device_context<platform::CUDADeviceContext>();
    platform::ForRange<platform::CUDADeviceContext> for_range(dev_ctx, numel);
    LgammaGradFunctorCUDA<T> functor(dout_data, x_data, dx_data, numel);
    for_range(functor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    lgamma, ops::LgammaKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LgammaKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    lgamma_grad,
    ops::LgammaGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LgammaGradKernel<paddle::platform::CUDADeviceContext, double>);
