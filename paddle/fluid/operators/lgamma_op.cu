#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <unsupported/Eigen/SpecialFunctions>
#include "paddle/fluid/operators/elementwise/elementwise_op_impl.cu.h"
#include "paddle/fluid/operators/lgamma_op.h"
#include "paddle/fluid/operators/math/complex_functors.h"

namespace paddle {
namespace operators {

template <typename T, typename Enable = void>
struct CudaLgammaFunctor;

template <typename T>
struct CudaLgammaFunctor<T, math::NoComplex<T, math::Real<T>>> {
  __device__ __forceinline__ T operator()(const T* args) const {
    return Eigen::numext::lgamma(args[0]);
  }
};

template <typename T>
class LgammaKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    const Tensor* x = context.Input<Tensor>("X");
    Tensor* out = context.Output<Tensor>("Out");
    out->mutable_data<math::Real<T>>(context.GetPlace());

    auto& dev_ctx = context.device_context<platform::CUDADeviceContext>();
    std::vector<const framework::Tensor*> ins = {x};
    std::vector<framework::Tensor*> outs = {out};
    auto functor = CudaLgammaFunctor<T>();
    LaunchSameDimsElementwiseCudaKernel<ElementwiseType::kUnary, T,
                                        math::Real<T>>(dev_ctx, ins, &outs,
                                                       functor);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    lgamma, ops::LgammaKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LgammaKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    lgamma_grad,
    ops::LgammaGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LgammaGradKernel<paddle::platform::CUDADeviceContext, double>);
