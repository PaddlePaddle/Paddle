#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hipblas.h>
#include <string>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/batch_fc_op.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {
using framework::Tensor;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
static inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__global__ void add_bias_kernel(T* data, int slot_pairs_num, int ins_num,
                                int out_dim, const T* bias) {
  CUDA_KERNEL_LOOP(idx, slot_pairs_num * ins_num * out_dim) {
    int block_len = ins_num * out_dim;
    int slot_index = idx / block_len;
    int out_dim_index = (idx % block_len) % out_dim;
    T temp = data[idx] + bias[slot_index * out_dim + out_dim_index];
    data[idx] = temp;
  }
}

template <typename T>
void add_bias(hipStream_t stream, T* data, int slot_pairs_num, int ins_num,
              int out_dim, const T* bias) {
  add_bias_kernel<<<GET_BLOCKS(slot_pairs_num * ins_num * out_dim),
                    CUDA_NUM_THREADS, 0, stream>>>(data, slot_pairs_num,
                                                   ins_num, out_dim, bias);
}

template <typename T>
__global__ void add_bias_grad_kernel(const T* dout_data, int slot_pairs_num,
                                     int ins_num, int out_dim, T* db_data) {
  CUDA_KERNEL_LOOP(idx, slot_pairs_num * out_dim) {
    int row = idx / out_dim;
    int col = idx % out_dim;
    T temp = static_cast<T>(0);
    for (int i = 0; i < ins_num; ++i) {
      int select_indx = ((row + 1) * i + 1) * col;
      temp += dout_data[select_indx];
    }
    db_data[idx] += temp;
  }
}

template <typename T>
void add_bias_grad(hipStream_t stream, const T* dout_data, int slot_pairs_num,
                   int ins_num, int out_dim, T* db_data) {
  add_bias_grad_kernel<<<GET_BLOCKS(slot_pairs_num * out_dim), CUDA_NUM_THREADS,
                         0, stream>>>(dout_data, slot_pairs_num, ins_num,
                                      out_dim, db_data);
}

template <typename DeviceContext, typename T>
class BatchFCCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    // X.dim = slot_pairs_num * ins_num * in_dim
    // W.dim = slot_pairs_num * in_dim * out_dim
    // b.dim = slot_pairs_num * out_dim
    // output.dim = slot_pairs_num * ins_num * out_dim
    auto* input = ctx.Input<framework::LoDTensor>("Input");
    auto* w = ctx.Input<Tensor>("W");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* output = ctx.Output<framework::LoDTensor>("Out");
    auto input_dims = input->dims();
    auto w_dims = w->dims();
    auto slot_pairs_num = input_dims[0];
    auto ins_num = input_dims[1];
    auto in_dim = input_dims[2];
    auto out_dim = w_dims[2];

    // get data ptr
    const T* in_data = input->data<T>();
    const T* w_data = w->data<T>();
    const T* bias_data = bias->data<T>();

    output->Resize({slot_pairs_num, ins_num, out_dim});
    T* out_data = output->mutable_data<T>(ctx.GetPlace());
    // initialize
    auto out_eigen = framework::EigenVector<T>::Flatten(*output);
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto& place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();
    out_eigen.device(place) = out_eigen.constant(static_cast<T>(0));

    CBLAS_TRANSPOSE transA = CblasNoTrans;
    CBLAS_TRANSPOSE transB = CblasNoTrans;

    T alpha = 1;
    T beta = 0;
    int64_t strideA = ins_num * in_dim;
    int64_t strideB = in_dim * out_dim;

    auto blas = math::GetBlas<platform::CUDADeviceContext, T>(dev_ctx);
    blas.BatchedGEMM(transA, transB, ins_num, out_dim, in_dim, alpha, in_data,
                     w_data, beta, out_data, slot_pairs_num, strideA, strideB);
    add_bias<T>(ctx.cuda_device_context().stream(), out_data, slot_pairs_num,
                ins_num, out_dim, bias_data);
  }
};

template <typename DeviceContext, typename T>
class BatchFCGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* input = ctx.Input<Tensor>("Input");
    auto* w = ctx.Input<Tensor>("W");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));

    auto* dx = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto* dw = ctx.Output<Tensor>(framework::GradVarName("W"));
    auto* db = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto input_dims = input->dims();
    auto w_dims = w->dims();
    auto slot_pairs_num = input_dims[0];
    auto ins_num = input_dims[1];
    auto in_dim = input_dims[2];
    auto out_dim = w_dims[2];

    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto& place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();
    // initialize
    dx->mutable_data<T>(ctx.GetPlace());
    auto dx_eigen = framework::EigenVector<T>::Flatten(*dx);
    dx_eigen.device(place) = dx_eigen.constant(static_cast<T>(0));

    dw->mutable_data<T>(ctx.GetPlace());
    auto dw_eigen = framework::EigenVector<T>::Flatten(*dw);
    dw_eigen.device(place) = dw_eigen.constant(static_cast<T>(0));

    // get data ptr
    const T* x_data = input->data<T>();
    const T* w_data = w->data<T>();
    const T* dout_data = dout->data<T>();
    T* dx_data = dx->data<T>();
    T* dw_data = dw->data<T>();

    db->mutable_data<T>(ctx.GetPlace());
    auto db_eigen = framework::EigenVector<T>::Flatten(*db);
    db_eigen.device(place) = db_eigen.constant(static_cast<T>(0));
    T* db_data = db->data<T>();
    add_bias_grad<T>(ctx.cuda_device_context().stream(), dout_data,
                     slot_pairs_num, ins_num, out_dim, db_data);

    auto blas = math::GetBlas<platform::CUDADeviceContext, T>(dev_ctx);
    T alpha = 1;
    T beta = 0;

    // dx = dout_data * y^T
    blas.BatchedGEMM(CblasNoTrans, CblasTrans, ins_num, in_dim, out_dim, alpha,
                     dout_data, w_data, beta, dx_data, slot_pairs_num,
                     ins_num * out_dim, out_dim * in_dim);
    // dy = x^T * dout_data
    blas.BatchedGEMM(CblasTrans, CblasNoTrans, in_dim, out_dim, ins_num, alpha,
                     x_data, dout_data, beta, dw_data, slot_pairs_num,
                     in_dim * ins_num, ins_num * out_dim);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using GPUCtx = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(batch_fc, ops::BatchFCCUDAKernel<GPUCtx, float>,
                        ops::BatchFCCUDAKernel<GPUCtx, double>);

REGISTER_OP_CUDA_KERNEL(batch_fc_grad,
                        ops::BatchFCGradOpCUDAKernel<GPUCtx, float>,
                        ops::BatchFCGradOpCUDAKernel<GPUCtx, double>);
