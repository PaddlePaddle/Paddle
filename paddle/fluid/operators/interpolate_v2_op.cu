#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <algorithm>
#include <string>
#include "paddle/fluid/operators/interpolate_v2_op.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace operators {

using framework::Tensor;
using DataLayout = framework::DataLayout;

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    if (data_layout == DataLayout::kNCHW) {
      out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idy * in_img_w + in_img_idx];
    } else {
      out[tid] = in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                    in_img_idx * num_channels + channel_id];
    }
  }
}

template <typename T>
__global__ void KeNearestNeighborInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                   in_img_idy * in_img_w + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                   in_img_idx * num_channels + channel_id];
    }
    const T out_pos = out[out_id_h * output_w + out_id_w];
    platform::CudaAtomicAdd(in_pos, out_pos);
  }
}

template <typename T>
__global__ void KeLinearInterpFw(const T* in, const size_t in_img_w,
                                 const size_t input_w, T* out,
                                 const size_t out_img_w, const size_t output_h,
                                 const size_t output_w,
                                 const size_t num_channels, const float ratio_w,
                                 const bool align_corners, const int align_mode,
                                 const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;  // w
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;  // w_id

    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos =
          &in[out_id_h * out_id_w + channel_id * in_img_size + in_img_idx];
      // linear interpolation
      out[out_id_h * output_w + out_id_w] =
          w2lambda * in_pos[0] + w1lambda * in_pos[w_id];

    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idx * num_channels + channel_id];
      // linear interpolation
      out[out_id_h * output_w + out_id_w] =
          w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels];
    }
  }
}

template <typename T>
__global__ void KeLinearInterpBw(T* in, const size_t in_img_w,
                                 const size_t input_w, const T* out,
                                 const size_t out_img_w, const size_t output_h,
                                 const size_t output_w,
                                 const size_t num_channels, const T ratio_w,
                                 const bool align_corners, const int align_mode,
                                 const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idx = align_flag ? ratio_w * (out_img_idx + 0.5) - 0.5
                                : ratio_w * out_img_idx;
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;  // w
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;  // w_id

    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    T* in_pos;
    if (data_layout == DataLayout::kNCHW) {
      in_pos = &in[out_id_h * input_w + channel_id * in_img_size + in_img_idx];
    } else {
      in_pos = &in[out_id_h * input_w + in_img_idx * num_channels + channel_id];
    }
    const T* out_pos = &out[out_id_w];

    if (data_layout == DataLayout::kNCHW) {
      platform::CudaAtomicAdd(&in_pos[0], w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id], w1lambda * out_pos[0]);
    } else {
      platform::CudaAtomicAdd(&in_pos[0], w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos[w_id * num_channels],
                              w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                            in_img_idy * in_img_w + in_img_idx];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                      w1lambda * in_pos[h_id * in_img_w + w_id]);
    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
              in_img_idx * num_channels + channel_id];

      // bilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          h2lambda *
              (w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w * num_channels] +
                      w1lambda * in_pos[h_id * in_img_w * num_channels +
                                        w_id * num_channels]);
    }
  }
}

template <typename T>
__forceinline__ __device__ void PreCalculatorForInputIndex(
    int* in_img_idx, int* in_img_idy, int* w_id, int* h_id, T* w1lambda,
    T* h1lambda, T* w2lambda, T* h2lambda, T src_w, T src_h, const int in_img_w,
    const int in_img_h) {
  src_w = (src_w > 0) ? src_w : 0.f;
  src_h = (src_h > 0) ? src_h : 0.f;
  *in_img_idx = static_cast<int>(src_w);
  *in_img_idy = static_cast<int>(src_h);
  *w_id = (*in_img_idx < in_img_w - 1) ? 1 : 0;
  *h_id = (*in_img_idy < in_img_h - 1) ? 1 : 0;
  *w1lambda = src_w - *in_img_idx;
  *h1lambda = src_h - *in_img_idy;
  *w2lambda = 1.f - *w1lambda;
  *h2lambda = 1.f - *h1lambda;
}

/* Calculate the minimum of partial elements in a block */
template <typename T>
__inline__ __device__ T PartialBlockMin(T val, size_t threads_num_in_block,
                                        unsigned mask) {
  __shared__ T shared[WARP_SIZE];
  __shared__ T shared_last_val;
  __shared__ int shared_last_idx;
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;
  int threshold = (threads_num_in_block & (-WARP_SIZE));

  if (threadIdx.x < threshold) {
    shared_last_idx = (threshold >> 5) - 1;
    val = math::warpReduceMin(val, mask);
    if (lane == 0) {
      shared[wid] = val;
    }
  } else {
    shared_last_val = std::numeric_limits<T>::max();
    platform::CudaAtomicMin(&shared_last_val, val);
    shared[wid] = shared_last_val;
    shared_last_idx = wid;
  }
  __syncthreads();

  if (threadIdx.x < threshold) {
    val = (lane <= shared_last_idx) ? shared[lane]
                                    : std::numeric_limits<T>::max();
    val = math::warpReduceMin(val, mask);
    shared_last_val = val;
  }
  __syncthreads();
  if (threadIdx.x >= threshold) {
    val = shared_last_val;
  }
  return val;
}

template <typename T>
__global__ void KeBilinearInterpBwShareMemory(
    T* in, const int in_h, const int in_w, const T* __restrict__ out,
    const int out_h, const int out_w, const int n, const int num_channels,
    float ratio_h, float ratio_w, const T align_type_value, bool is_nchw) {
  __shared__ T s_data[2][1024];
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int in_chw = in_h * in_w * num_channels;
  int out_chw = num_channels * out_h * out_w;
  int nthreads = n * out_chw;

  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / out_chw;
    int out_id_w = tid % out_chw;
    const int in_img_size = in_h * in_w;
    const int out_img_size = out_h * out_w;
    T value = out[out_id_h * out_chw + out_id_w];

    int channel_id = out_id_w / out_img_size;
    int out_img_idy = (out_id_w % out_img_size) / out_w;
    int out_img_idx = tid % out_w;

    int in_img_idx, in_img_idy, w_id, h_id;
    T w1lambda, h1lambda, w2lambda, h2lambda;
    T src_w = ratio_w * (out_img_idx + align_type_value) - align_type_value;
    T src_h = ratio_h * (out_img_idy + align_type_value) - align_type_value;
    PreCalculatorForInputIndex(&in_img_idx, &in_img_idy, &w_id, &h_id,
                               &w1lambda, &h1lambda, &w2lambda, &h2lambda,
                               src_w, src_h, in_w, in_h);

    // top_left_index is just input_index.
    int input_index = out_id_h * in_chw + channel_id * in_img_size +
                      in_img_idy * in_w + in_img_idx;
    int top_right_index = input_index + w_id;
    int bot_left_index = input_index + h_id * in_w;
    int bot_right_index = input_index + h_id * in_w + w_id;
    int in_top_min_index, in_bot_min_index;

    s_data[0][threadIdx.x] = 0.f;
    s_data[1][threadIdx.x] = 0.f;
    int remain = nthreads - (tid & (-blockDim.x));
    int in_top_max_index = math::blockReduceMax(top_right_index, FINAL_MASK);
    int in_bot_max_index = math::blockReduceMax(bot_right_index, FINAL_MASK);

    if (remain > blockDim.x) {
      in_top_min_index = math::blockReduceMin(input_index, FINAL_MASK);
      in_bot_min_index = math::blockReduceMin(bot_left_index, FINAL_MASK);
    } else {
      in_top_min_index = PartialBlockMin(input_index, remain, FINAL_MASK);
      in_bot_min_index = PartialBlockMin(bot_left_index, remain, FINAL_MASK);
    }
    int upper_limit_share_idx = (in_top_max_index - in_top_min_index) >
                                        (in_bot_max_index - in_bot_min_index)
                                    ? (in_top_max_index - in_top_min_index)
                                    : (in_bot_max_index - in_bot_min_index);
    if (h_id != 0) {
      platform::CudaAtomicAdd(&s_data[0][input_index - in_top_min_index],
                              h2lambda * w2lambda * value);
      platform::CudaAtomicAdd(&s_data[0][top_right_index - in_top_min_index],
                              h2lambda * w1lambda * value);
      platform::CudaAtomicAdd(&s_data[1][bot_left_index - in_bot_min_index],
                              h1lambda * w2lambda * value);
      platform::CudaAtomicAdd(&s_data[1][bot_right_index - in_bot_min_index],
                              h1lambda * w1lambda * value);
    } else {
      platform::CudaAtomicAdd(&s_data[0][top_right_index - in_top_min_index],
                              (h2lambda + h1lambda) * w1lambda * value);
      platform::CudaAtomicAdd(&s_data[1][bot_left_index - in_bot_min_index],
                              (h1lambda + h2lambda) * w2lambda * value);
    }
    __syncthreads();

    if (threadIdx.x <= upper_limit_share_idx) {
      platform::CudaAtomicAdd(&in[in_top_min_index + threadIdx.x],
                              s_data[0][threadIdx.x]);
      platform::CudaAtomicAdd(&in[in_bot_min_index + threadIdx.x],
                              s_data[1][threadIdx.x]);
    }
  }
}

template <typename T>
__global__ void KeBilinearInterpBw(T* in, const int in_h, const int in_w,
                                   const T* __restrict__ out, const int out_h,
                                   const int out_w, const int n,
                                   const int num_channels, float ratio_h,
                                   float ratio_w, const T align_type_value,
                                   bool is_nchw) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int in_chw = in_h * in_w * num_channels;
  int out_chw = num_channels * out_h * out_w;
  int nthreads = n * out_chw;

  if (is_nchw) {
    for (; tid < nthreads; tid += stride) {
      int out_id_h = tid / out_chw;
      int out_id_w = tid % out_chw;
      const int in_img_size = in_h * in_w;
      const int out_img_size = out_h * out_w;
      T value = out[out_id_h * out_chw + out_id_w];

      int channel_id = out_id_w / out_img_size;
      int out_img_idy = (out_id_w % out_img_size) / out_w;
      int out_img_idx = tid % out_w;
      int in_img_idx, in_img_idy, w_id, h_id;
      T w1lambda, h1lambda, w2lambda, h2lambda;

      T src_w = ratio_w * (out_img_idx + align_type_value) - align_type_value;
      T src_h = ratio_h * (out_img_idy + align_type_value) - align_type_value;
      PreCalculatorForInputIndex(&in_img_idx, &in_img_idy, &w_id, &h_id,
                                 &w1lambda, &h1lambda, &w2lambda, &h2lambda,
                                 src_w, src_h, in_w, in_h);

      T* in_pos = &in[out_id_h * in_chw + channel_id * in_img_size +
                      in_img_idy * in_w + in_img_idx];
      platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * value);
      platform::CudaAtomicAdd(&in_pos[w_id], h2lambda * w1lambda * value);
      platform::CudaAtomicAdd(&in_pos[h_id * in_w],
                              h1lambda * w2lambda * value);
      platform::CudaAtomicAdd(&in_pos[h_id * in_w + w_id],
                              h1lambda * w1lambda * value);
    }
  } else {
    for (; tid < nthreads; tid += stride) {
      int out_id_h = tid / out_chw;
      int out_id_w = tid % out_chw;
      const int in_img_size = in_h * in_w;
      const int out_img_size = out_h * out_w;
      T value = out[out_id_h * out_chw + out_id_w];

      int out_img_idy = out_id_w / (out_w * num_channels);
      int out_img_idx = out_id_w % (out_w * num_channels) / num_channels;
      int channel_id = tid % num_channels;

      int in_img_idx, in_img_idy, w_id, h_id;
      T w1lambda, h1lambda, w2lambda, h2lambda;
      T src_w = ratio_w * (out_img_idx + align_type_value) - align_type_value;
      T src_h = ratio_h * (out_img_idy + align_type_value) - align_type_value;
      PreCalculatorForInputIndex(&in_img_idx, &in_img_idy, &w_id, &h_id,
                                 &w1lambda, &h1lambda, &w2lambda, &h2lambda,
                                 src_w, src_h, in_w, in_h);

      T* in_pos = &in[out_id_h * in_chw + in_img_idy * in_w * num_channels +
                      in_img_idx * num_channels + channel_id];
      platform::CudaAtomicAdd(&in_pos[0], h2lambda * w2lambda * value);
      platform::CudaAtomicAdd(&in_pos[w_id * num_channels],
                              h2lambda * w1lambda * value);
      platform::CudaAtomicAdd(&in_pos[h_id * in_w * num_channels],
                              h1lambda * w2lambda * value);
      platform::CudaAtomicAdd(
          &in_pos[h_id * in_w * num_channels + w_id * num_channels],
          h1lambda * w1lambda * value);
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpFw(
    const T* in, const size_t in_img_d, const size_t in_img_h,
    const size_t in_img_w, const size_t input_h, const size_t input_w, T* out,
    const size_t out_img_d, const size_t out_img_h, const size_t out_img_w,
    const size_t output_h, const size_t output_w, const size_t num_channels,
    const float ratio_d, const float ratio_h, const float ratio_w,
    const bool align_corners, const int align_mode,
    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    T src_d = ratio_d * (out_img_idt + 0.5) - 0.5;
    src_d = (src_d > 0) ? src_d : 0;
    T d1lambda =
        align_flag ? src_d - in_img_idt : ratio_d * out_img_idt - in_img_idt;
    T d2lambda = 1.f - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] + w1lambda * in_pos1[w_id]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w] +
                           w1lambda * in_pos1[h_id * in_img_w + w_id])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] + w1lambda * in_pos2[w_id]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w] +
                           w1lambda * in_pos2[h_id * in_img_w + w_id]));

    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] +
                           w1lambda * in_pos1[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos1[h_id * in_img_w * num_channels +
                                              w_id * num_channels])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] +
                           w1lambda * in_pos2[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos2[h_id * in_img_w * num_channels +
                                              w_id * num_channels]));
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpBw(
    T* in, const size_t in_img_d, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, const T* out,
    const size_t out_img_d, const size_t out_img_h, const size_t out_img_w,
    const size_t output_h, const size_t output_w, const size_t num_channels,
    const T ratio_d, const T ratio_h, const T ratio_w, const bool align_corners,
    const int align_mode, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    T src_d = ratio_d * (out_img_idt + 0.5) - 0.5;
    src_d = (src_d > 0) ? src_d : 0;
    T d1lambda =
        align_flag ? src_d - in_img_idt : ratio_d * out_img_idt - in_img_idt;
    T d2lambda = 1.f - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = ratio_h * (out_img_idy + 0.5) - 0.5;
    src_h = (src_h > 0) ? src_h : 0;
    T h1lambda =
        align_flag ? src_h - in_img_idy : ratio_h * out_img_idy - in_img_idy;
    T h2lambda = 1.f - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = ratio_w * (out_img_idx + 0.5) - 0.5;
    src_w = (src_w > 0) ? src_w : 0;
    T w1lambda =
        align_flag ? src_w - in_img_idx : ratio_w * out_img_idx - in_img_idx;
    T w2lambda = 1.f - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      T* in_pos2 = &in[in_pos2_idx];

      const T* out_pos = &out[out_id_h * output_w + out_id_w];

      // trilinear interpolation grad
      platform::CudaAtomicAdd(&in_pos1[0],
                              d2lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[w_id],
                              d2lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w],
                              d2lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w + w_id],
                              d2lambda * h1lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[0],
                              d1lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[w_id],
                              d1lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w],
                              d1lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w + w_id],
                              d1lambda * h1lambda * w1lambda * out_pos[0]);
    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      T* in_pos2 = &in[in_pos2_idx];

      const T* out_pos = &out[out_id_h * output_w + out_id_w];

      // trilinear interpolation grad
      platform::CudaAtomicAdd(&in_pos1[0],
                              d2lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[w_id * num_channels],
                              d2lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos1[h_id * in_img_w * num_channels],
                              d2lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos1[h_id * in_img_w * num_channels + w_id * num_channels],
          d2lambda * h1lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[0],
                              d1lambda * h2lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[w_id * num_channels],
                              d1lambda * h2lambda * w1lambda * out_pos[0]);
      platform::CudaAtomicAdd(&in_pos2[h_id * in_img_w * num_channels],
                              d1lambda * h1lambda * w2lambda * out_pos[0]);
      platform::CudaAtomicAdd(
          &in_pos2[h_id * in_img_w * num_channels + w_id * num_channels],
          d1lambda * h1lambda * w1lambda * out_pos[0]);
    }
  }
}

template <typename T>
__device__ __forceinline__ static T Kecubic_interp(const T x0, const T x1,
                                                   const T x2, const T x3,
                                                   T t) {
  T coeffs[4];
  T a = -0.75;
  T x_1 = t;
  T x_2 = 1.0 - t;
  coeffs[0] = cubic_convolution2<T>(x_1 + 1.0, a);
  coeffs[1] = cubic_convolution1<T>(x_1, a);
  coeffs[2] = cubic_convolution1<T>(x_2, a);
  coeffs[3] = cubic_convolution2<T>(x_2 + 1.0, a);
  return x0 * coeffs[0] + x1 * coeffs[1] + x2 * coeffs[2] + x3 * coeffs[3];
}

template <typename T>
__global__ void KeBicubicInterpFw(
    const T* in, const size_t in_img_h, const size_t in_img_w,
    const size_t input_h, const size_t input_w, T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;

    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    T in_img_idy = align_corners
                       ? static_cast<T>(ratio_h * out_img_idy)
                       : static_cast<T>(ratio_h * (out_img_idy + 0.5) - 0.5);
    int input_y = floorf(in_img_idy);
    const T y_t = in_img_idy - input_y;

    T in_img_idx = align_corners
                       ? static_cast<T>(ratio_w * out_img_idx)
                       : static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    int input_x = floorf(in_img_idx);
    const T x_t = in_img_idx - input_x;

    T coefficients[4];
    const T* in_pos_0;
    const T* in_pos_1;
    const T* in_pos_2;
    const T* in_pos_3;
    int access_x_0;
    if (data_layout == DataLayout::kNCHW) {
      for (int k = 0; k < 4; k++) {
        int access_y =
            max(min(input_y - 1 + k, static_cast<int>(in_img_h - 1)), 0);
        access_x_0 = max(min(input_x - 1, static_cast<int>(in_img_w - 1)), 0);
        int access_x_1 =
            max(min(input_x + 0, static_cast<int>(in_img_w - 1)), 0);
        int access_x_2 =
            max(min(input_x + 1, static_cast<int>(in_img_w - 1)), 0);
        int access_x_3 =
            max(min(input_x + 2, static_cast<int>(in_img_w - 1)), 0);

        in_pos_0 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_0];
        in_pos_1 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_1];
        in_pos_2 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_2];
        in_pos_3 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_3];

        coefficients[k] = Kecubic_interp<T>(in_pos_0[0], in_pos_1[0],
                                            in_pos_2[0], in_pos_3[0], x_t);
      }

      out[out_id_h * output_w + out_id_w] =
          Kecubic_interp<T>(coefficients[0], coefficients[1], coefficients[2],
                            coefficients[3], y_t);

    } else {
      for (int k = 0; k < 4; k++) {
        int access_y =
            max(min(input_y - 1 + k, static_cast<int>((in_img_h - 1))), 0);
        int access_x_0 =
            max(min(input_x - 1, static_cast<int>((in_img_w - 1))), 0);
        int access_x_1 =
            max(min(input_x + 0, static_cast<int>((in_img_w - 1))), 0);
        int access_x_2 =
            max(min(input_x + 1, static_cast<int>((in_img_w - 1))), 0);
        int access_x_3 =
            max(min(input_x + 2, static_cast<int>((in_img_w - 1))), 0);

        const T* in_pos_0 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_0 * num_channels + channel_id];
        const T* in_pos_1 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_1 * num_channels + channel_id];
        const T* in_pos_2 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_2 * num_channels + channel_id];
        const T* in_pos_3 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_3 * num_channels + channel_id];

        coefficients[k] = Kecubic_interp(in_pos_0[0], in_pos_1[0], in_pos_2[0],
                                         in_pos_3[0], x_t);
      }

      out[out_id_h * output_w + out_id_w] =
          static_cast<T>(Kecubic_interp(coefficients[0], coefficients[1],
                                        coefficients[2], coefficients[3], y_t));
    }
  }
}

template <typename T>
__global__ void KeBicubicInterpBw(
    T* in, const size_t in_img_h, const size_t in_img_w, const size_t input_h,
    const size_t input_w, const T* out, const size_t out_img_h,
    const size_t out_img_w, const size_t output_h, const size_t output_w,
    const size_t num_channels, const float ratio_h, const float ratio_w,
    const bool align_corners, const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    T in_img_idy = align_corners
                       ? static_cast<T>(ratio_h * out_img_idy)
                       : static_cast<T>(ratio_h * (out_img_idy + 0.5) - 0.5);
    int input_y = floorf(in_img_idy);
    const T y_t = in_img_idy - input_y;

    T in_img_idx = align_corners
                       ? static_cast<T>(ratio_w * out_img_idx)
                       : static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    int input_x = floorf(in_img_idx);

    const T x_t = in_img_idx - input_x;

    T x_coeffs[4];
    T y_coeffs[4];

    get_cubic_upsample_coefficients(x_coeffs, x_t);
    get_cubic_upsample_coefficients(y_coeffs, y_t);

    const T* out_pos = &out[out_id_h * output_w + out_id_w];
    T* in_pos;

    for (int i = 0; i < 4; i++) {
      for (int j = 0; j < 4; j++) {
        int access_y = max(min(static_cast<int>(input_y - 1 + j),
                               static_cast<int>(in_img_h - 1)),
                           0);
        int access_x = max(min(static_cast<int>(input_x - 1 + i),
                               static_cast<int>(in_img_w - 1)),
                           0);
        if (data_layout == DataLayout::kNCHW) {
          in_pos = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x];
        } else {
          in_pos = &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                       access_x * num_channels + channel_id];
        }
        platform::CudaAtomicAdd(&in_pos[0],
                                (out_pos[0] * y_coeffs[j] * x_coeffs[i]));
      }
    }
  }
}

template <typename T>
static void Interpolate1DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  float scale_w = -1;
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_w = new_size[0];
  } else {
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    auto scale = ctx.Attr<std::vector<float>>("scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      scale_w = scale_data[0];
      PADDLE_ENFORCE_EQ(
          scale_w > 0, true,
          platform::errors::InvalidArgument(
              "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
    } else {
      if (scale.size() > 0) {
        scale_w = scale[0];
        PADDLE_ENFORCE_EQ(
            scale_w > 0, true,
            platform::errors::InvalidArgument(
                "The scale_w in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_w));
      }
    }
    if (scale_w > 0.) {
      out_w = static_cast<int>(in_w * scale_w);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_w = size_data[0];
    }
  }
  PADDLE_ENFORCE_GT(out_w, 0, platform::errors::InvalidArgument(
                                  "out_w in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_w};
  } else {
    dim_out = {n, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_w = 0.f;
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1.0) / (out_w - 1.0)
                              : static_cast<float>(new_scale_w);
  }

  int in_cw = c * in_w;
  int out_cw = c * out_w;
  int pixelNum = n * out_cw;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

  if ("linear" == interp_method) {
    KeLinearInterpFw<T><<<config.block_per_grid, config.thread_per_block, 0,
                          ctx.cuda_device_context().stream()>>>(
        input_data, in_w, in_cw, output_data, out_w, n, out_cw, c, ratio_w,
        align_corners, align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate2DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  float scale_w = -1;
  float scale_h = -1;
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_h = new_size[0];
    out_w = new_size[1];
  } else {
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    auto scale = ctx.Attr<std::vector<float>>("scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      if (scale_data.size() > 1) {
        scale_h = scale_data[0];
        scale_w = scale_data[1];
      } else {
        scale_h = scale_data[0];
        scale_w = scale_data[0];
      }

      PADDLE_ENFORCE_EQ(
          scale_w > 0, true,
          platform::errors::InvalidArgument(
              "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
      PADDLE_ENFORCE_EQ(
          scale_h > 0, true,
          platform::errors::InvalidArgument(
              "The scale_h in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_h));
    } else {
      if (scale.size() > 1) {
        scale_w = scale[1];
        scale_h = scale[0];

        PADDLE_ENFORCE_EQ(
            scale_w > 0, true,
            platform::errors::InvalidArgument(
                "The scale_w in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_w));
        PADDLE_ENFORCE_EQ(
            scale_h > 0, true,
            platform::errors::InvalidArgument(
                "The scale_h in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_h));
      }
    }
    if (scale_w > 0. && scale_h > 0.) {
      out_h = static_cast<int>(in_h * scale_h);
      out_w = static_cast<int>(in_w * scale_w);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }
  }
  PADDLE_ENFORCE_GT(out_h, 0, platform::errors::InvalidArgument(
                                  "out_h in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  PADDLE_ENFORCE_GT(out_w, 0, platform::errors::InvalidArgument(
                                  "out_w in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_h, out_w};
  } else {
    dim_out = {n, out_h, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_h == out_h && in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    float new_scale_h = 0.f;
    new_scale_h = (scale_h > 0) ? static_cast<float>(1. / scale_h)
                                : static_cast<float>(in_h) / out_h;
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(new_scale_h);
  }
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;

  int pixelNum = n * out_chw;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpFw<
        T><<<config.block_per_grid, config.thread_per_block, 0,
             ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    KeBilinearInterpFw<T><<<config.block_per_grid, config.thread_per_block, 0,
                            ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, align_mode, data_layout);
  } else if ("bicubic" == interp_method) {
    KeBicubicInterpFw<T><<<config.block_per_grid, 512, 0,
                           ctx.cuda_device_context().stream()>>>(
        input_data, in_h, in_w, n, in_chw, output_data, out_h, out_w, n,
        out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  }
}

template <typename T>
static void Interpolate3DCUDAFwd(const framework::ExecutionContext& ctx,
                                 const Tensor& input, Tensor* output) {
  auto* input_data = input.data<T>();

  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_d = ctx.Attr<int>("out_d");
  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");

  auto list_new_shape_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  float scale_w = -1;
  float scale_d = -1;
  float scale_h = -1;
  if (list_new_shape_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_shape_tensor);
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  } else {
    auto scale_tensor = ctx.Input<Tensor>("Scale");
    auto scale = ctx.Attr<std::vector<float>>("scale");
    if (scale_tensor != nullptr) {
      auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
      if (scale_data.size() > 1) {
        scale_d = scale_data[0];
        scale_h = scale_data[1];
        scale_w = scale_data[2];
      } else {
        scale_d = scale_data[0];
        scale_h = scale_data[0];
        scale_w = scale_data[0];
      }

      PADDLE_ENFORCE_EQ(
          scale_w > 0, true,
          platform::errors::InvalidArgument(
              "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
      PADDLE_ENFORCE_EQ(
          scale_h > 0, true,
          platform::errors::InvalidArgument(
              "The scale_h in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_h));
      PADDLE_ENFORCE_EQ(
          scale_d > 0, true,
          platform::errors::InvalidArgument(
              "The scale_d in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_d));
    } else {
      if (scale.size() > 1) {
        scale_d = scale[0];
        scale_h = scale[1];
        scale_w = scale[2];

        PADDLE_ENFORCE_EQ(
            scale_w > 0, true,
            platform::errors::InvalidArgument(
                "The scale_w in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_w));
        PADDLE_ENFORCE_EQ(
            scale_h > 0, true,
            platform::errors::InvalidArgument(
                "The scale_h in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_h));
        PADDLE_ENFORCE_EQ(
            scale_d > 0, true,
            platform::errors::InvalidArgument(
                "The scale_d in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_d));
      }
    }
    if (scale_d > 0. && scale_h > 0. && scale_w > 0.) {
      out_d = static_cast<int>(in_d * scale_d);
      out_h = static_cast<int>(in_h * scale_h);
      out_w = static_cast<int>(in_w * scale_w);
    }
    auto out_size = ctx.Input<Tensor>("OutSize");
    if (out_size != nullptr) {
      Tensor sizes;
      framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_d = size_data[0];
      out_h = size_data[1];
      out_w = size_data[2];
    }
  }
  PADDLE_ENFORCE_GT(out_d, 0, platform::errors::InvalidArgument(
                                  "out_d in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  PADDLE_ENFORCE_GT(out_h, 0, platform::errors::InvalidArgument(
                                  "out_h in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));
  PADDLE_ENFORCE_GT(out_w, 0, platform::errors::InvalidArgument(
                                  "out_w in Attr(out_shape) of Op(interpolate) "
                                  "should be greater than 0."));

  framework::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_d, out_h, out_w};
  } else {
    dim_out = {n, out_d, out_h, out_w, c};
  }
  auto output_data = output->mutable_data<T>(dim_out, ctx.GetPlace());

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    framework::TensorCopy(input, ctx.GetPlace(), output);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    float new_scale_d = 0.f;
    new_scale_d = (scale_d > 0) ? static_cast<float>(1. / scale_d)
                                : static_cast<float>(in_d) / out_d;
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(new_scale_d);
  }
  if (out_h > 1) {
    float new_scale_h = 0.f;
    new_scale_h = (scale_h > 0) ? static_cast<float>(1. / scale_h)
                                : static_cast<float>(in_h) / out_h;
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(new_scale_h);
  }
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }

  int in_dhw = in_d * in_h * in_w;
  int out_dhw = out_d * out_h * out_w;
  int in_cdhw = c * in_dhw;
  int out_cdhw = c * out_dhw;

  int pixelNum = n * out_cdhw;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

  if ("trilinear" == interp_method) {
    KeTrilinearInterpFw<T><<<config.block_per_grid, config.thread_per_block, 0,
                             ctx.cuda_device_context().stream()>>>(
        input_data, in_d, in_h, in_w, n, in_cdhw, output_data, out_d, out_h,
        out_w, n, out_cdhw, c, ratio_d, ratio_h, ratio_w, align_corners,
        align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate1DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad, const Tensor output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_w = ctx.Attr<int>("out_w");
  float scale_w = -1;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  auto scale = ctx.Attr<std::vector<float>>("scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    scale_w = scale_data[0];
    PADDLE_ENFORCE_EQ(
        scale_w > 0, true,
        platform::errors::InvalidArgument(
            "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
            "should be greater than 0, but received value is %d.",
            scale_w));
  } else {
    if (scale.size() > 0) {
      scale_w = scale[0];

      PADDLE_ENFORCE_EQ(
          scale_w > 0, true,
          platform::errors::InvalidArgument(
              "The scale_w in Attr(scale) of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
    }
  }
  if (scale_w > 0.) {
    out_w = static_cast<int>(in_w * scale_w);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_w = size_data[0];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_w = new_size[0];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_w};
  } else {
    dim_grad = {n, in_w, c};
  }
  input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_w = 0.f;
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }
  int in_cw = c * in_w;
  int out_cw = c * out_w;
  int pixelNum = n * out_cw;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

  if ("linear" == interp_method) {
    KeLinearInterpBw<T><<<config.block_per_grid, config.thread_per_block, 0,
                          ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_w, in_cw, output_grad_data, out_w, n, out_cw, c,
        ratio_w, align_corners, align_mode, data_layout);
  }
}

template <typename T>
static void Interpolate2DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad, const Tensor output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");
  float scale_h = -1;
  float scale_w = -1;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  auto scale = ctx.Attr<std::vector<float>>("scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    if (scale_data.size() > 1) {
      scale_h = scale_data[0];
      scale_w = scale_data[1];
    } else {
      scale_h = scale_data[0];
      scale_w = scale_data[0];
    }

    PADDLE_ENFORCE_EQ(
        scale_w > 0, true,
        platform::errors::InvalidArgument(
            "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
            "should be greater than 0, but received value is %d.",
            scale_w));
    PADDLE_ENFORCE_EQ(
        scale_h > 0, true,
        platform::errors::InvalidArgument(
            "The scale_h in input 'Scale' Tensor of Operator(interpolate) "
            "should be greater than 0, but received value is %d.",
            scale_h));
  } else {
    if (scale.size() > 1) {
      scale_w = scale[1];
      scale_h = scale[0];

      PADDLE_ENFORCE_EQ(
          scale_w > 0, true,
          platform::errors::InvalidArgument(
              "The scale_w in Attr(scale) of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
      PADDLE_ENFORCE_EQ(
          scale_h > 0, true,
          platform::errors::InvalidArgument(
              "The scale_h in Attr(scale) of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_h));
    }
  }
  if (scale_w > 0. && scale_h > 0.) {
    out_h = static_cast<int>(in_h * scale_h);
    out_w = static_cast<int>(in_w * scale_w);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_h = size_data[0];
    out_w = size_data[1];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_h = new_size[0];
    out_w = new_size[1];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_h, in_w};
  } else {
    dim_grad = {n, in_h, in_w, c};
  }
  input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_h == out_h && in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    float new_scale_h = 0.f;
    new_scale_h = (scale_h > 0) ? static_cast<float>(1. / scale_h)
                                : static_cast<float>(in_h) / out_h;
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(new_scale_h);
  }
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }

  int in_hw = in_h * in_w;
  int out_hw = out_h * out_w;
  int in_chw = c * in_hw;
  int out_chw = c * out_hw;
  int pixelNum = n * out_chw;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

  if ("nearest" == interp_method) {
    KeNearestNeighborInterpBw<
        T><<<config.block_per_grid, config.thread_per_block, 0,
             ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  } else if ("bilinear" == interp_method) {
    const T align_type_value = (align_mode == 0 && !align_corners) ? 0.5f : 0;
    bool is_nchw = (data_layout == DataLayout::kNCHW) ? true : false;
    bool optimize_flag = false;
    optimize_flag = (in_h < (out_h >> 6) && in_w < (out_w >> 6))
                        ? true
                        : ((in_h == 1 && in_w == 1) ? true : false);

    if (optimize_flag & is_nchw) {
      KeBilinearInterpBwShareMemory<
          T><<<config.block_per_grid, config.thread_per_block, 0,
               ctx.cuda_device_context().stream()>>>(
          input_grad_data, in_h, in_w, output_grad_data, out_h, out_w, n, c,
          ratio_h, ratio_w, align_type_value, is_nchw);
    } else {
      KeBilinearInterpBw<T><<<config.block_per_grid, config.thread_per_block, 0,
                              ctx.cuda_device_context().stream()>>>(
          input_grad_data, in_h, in_w, output_grad_data, out_h, out_w, n, c,
          ratio_h, ratio_w, align_type_value, is_nchw);
    }
  } else if ("bicubic" == interp_method) {
    KeBicubicInterpBw<T><<<config.block_per_grid, 512, 0,
                           ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_h, in_w, n, in_chw, output_grad_data, out_h, out_w,
        n, out_chw, c, ratio_h, ratio_w, align_corners, data_layout);
  }
}

template <typename T>
static void Interpolate3DCUDABwd(const framework::ExecutionContext& ctx,
                                 Tensor* input_grad,
                                 const Tensor& output_grad) {
  auto* input = ctx.Input<Tensor>("X");
  const std::string data_layout_str = ctx.Attr<std::string>("data_layout");
  const DataLayout data_layout = framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  ExtractNCDWH(input->dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  auto interp_method = ctx.Attr<std::string>("interp_method");
  bool align_corners = ctx.Attr<bool>("align_corners");
  int align_mode = ctx.Attr<int>("align_mode");

  int out_d = ctx.Attr<int>("out_d");
  int out_h = ctx.Attr<int>("out_h");
  int out_w = ctx.Attr<int>("out_w");
  float scale_d = -1;
  float scale_h = -1;
  float scale_w = -1;
  auto scale_tensor = ctx.Input<Tensor>("Scale");
  auto scale = ctx.Attr<std::vector<float>>("scale");
  if (scale_tensor != nullptr) {
    auto scale_data = get_new_data_from_tensor<float>(scale_tensor);
    if (scale_data.size() > 1) {
      scale_d = scale_data[0];
      scale_h = scale_data[1];
      scale_w = scale_data[2];
    } else {
      scale_d = scale_data[0];
      scale_h = scale_data[0];
      scale_w = scale_data[0];
    }
    PADDLE_ENFORCE_EQ(
        scale_w > 0, true,
        platform::errors::InvalidArgument(
            "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
            "should be greater than 0, but received value is %d.",
            scale_w));
    PADDLE_ENFORCE_EQ(
        scale_h > 0, true,
        platform::errors::InvalidArgument(
            "The scale_h in input 'Scale' Tensor of Operator(interpolate) "
            "should be greater than 0, but received value is %d.",
            scale_h));
    PADDLE_ENFORCE_EQ(
        scale_d > 0, true,
        platform::errors::InvalidArgument(
            "The scale_d in input 'Scale' Tensor of Operator(interpolate) "
            "should be greater than 0, but received value is %d.",
            scale_d));
  } else {
    if (scale.size() > 1) {
      scale_d = scale[0];
      scale_h = scale[1];
      scale_w = scale[2];

      PADDLE_ENFORCE_EQ(
          scale_w > 0, true,
          platform::errors::InvalidArgument(
              "The scale_w in Attr(scale) of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
      PADDLE_ENFORCE_EQ(
          scale_h > 0, true,
          platform::errors::InvalidArgument(
              "The scale_h in Attr(scale) of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_h));
      PADDLE_ENFORCE_EQ(
          scale_d > 0, true,
          platform::errors::InvalidArgument(
              "The scale_d in Attr(scale) of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_d));
    }
  }
  if (scale_d > 0. && scale_h > 0. && scale_w > 0.) {
    out_d = static_cast<int>(in_d * scale_d);
    out_h = static_cast<int>(in_h * scale_h);
    out_w = static_cast<int>(in_w * scale_w);
  }

  auto out_size = ctx.Input<Tensor>("OutSize");
  if (out_size != nullptr) {
    Tensor sizes;
    framework::TensorCopySync(*out_size, platform::CPUPlace(), &sizes);
    auto size_data = sizes.data<int>();
    out_d = size_data[0];
    out_h = size_data[1];
    out_w = size_data[2];
  }
  auto list_new_size_tensor = ctx.MultiInput<framework::Tensor>("SizeTensor");
  if (list_new_size_tensor.size() > 0) {
    // have size tensor
    auto new_size = get_new_shape(list_new_size_tensor);
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  }

  auto* output_grad_data = output_grad.data<T>();
  framework::DDim dim_grad;
  if (data_layout == DataLayout::kNCHW) {
    dim_grad = {n, c, in_d, in_h, in_w};
  } else {
    dim_grad = {n, in_d, in_h, in_w, c};
  }
  auto* input_grad_data = input_grad->mutable_data<T>(dim_grad, ctx.GetPlace());
  auto& device_ctx = ctx.template device_context<platform::CUDADeviceContext>();
  math::SetConstant<platform::CUDADeviceContext, T> zero;
  zero(device_ctx, input_grad, static_cast<T>(0.0));

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    framework::TensorCopy(output_grad, ctx.GetPlace(), input_grad);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    float new_scale_d = 0.f;
    new_scale_d = (scale_d > 0) ? static_cast<float>(1. / scale_d)
                                : static_cast<float>(in_d) / out_d;
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(new_scale_d);
  }
  if (out_h > 1) {
    float new_scale_h = 0.f;
    new_scale_h = (scale_h > 0) ? static_cast<float>(1. / scale_h)
                                : static_cast<float>(in_h) / out_h;
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(new_scale_h);
  }
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }

  int in_dhw = in_d * in_h * in_w;
  int out_dhw = out_d * out_h * out_w;
  int in_cdhw = c * in_dhw;
  int out_cdhw = c * out_dhw;

  int pixelNum = n * out_cdhw;

  platform::GpuLaunchConfig config =
      platform::GetGpuLaunchConfig1D(ctx.cuda_device_context(), pixelNum);

  if ("trilinear" == interp_method) {
    KeTrilinearInterpBw<T><<<config.block_per_grid, config.thread_per_block, 0,
                             ctx.cuda_device_context().stream()>>>(
        input_grad_data, in_d, in_h, in_w, n, in_cdhw, output_grad_data, out_d,
        out_h, out_w, n, out_cdhw, c, ratio_d, ratio_h, ratio_w, align_corners,
        align_mode, data_layout);
  }
}

template <typename T>
class InterpolateOpV2CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");

    auto input_dims = input->dims();
    if (input_dims.size() == 3) {  // 1D interpolation
      Interpolate1DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDAFwd<T>(ctx, *input, output);
    } else if (input_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDAFwd<T>(ctx, *input, output);
    }
  }
};

template <typename T>
class InterpolateV2GradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::NotFound("This kernel only runs on GPU device."));
    auto* input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto* output_grad = ctx.Input<Tensor>(framework::GradVarName("Out"));

    auto output_grad_dims = output_grad->dims();
    if (output_grad_dims.size() == 3) {  // 1D interpolation
      Interpolate1DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 4) {  // 2D interpolation
      Interpolate2DCUDABwd<T>(ctx, input_grad, *output_grad);
    } else if (output_grad_dims.size() == 5) {  // 3D interpolation
      Interpolate3DCUDABwd<T>(ctx, input_grad, *output_grad);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(bilinear_interp_v2,
                        ops::InterpolateOpV2CUDAKernel<float>,
                        ops::InterpolateOpV2CUDAKernel<double>,
                        ops::InterpolateOpV2CUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bilinear_interp_v2_grad,
                        ops::InterpolateV2GradOpCUDAKernel<float>,
                        ops::InterpolateV2GradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(nearest_interp_v2,
                        ops::InterpolateOpV2CUDAKernel<float>,
                        ops::InterpolateOpV2CUDAKernel<double>,
                        ops::InterpolateOpV2CUDAKernel<int64_t>,
                        ops::InterpolateOpV2CUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(nearest_interp_v2_grad,
                        ops::InterpolateV2GradOpCUDAKernel<float>,
                        ops::InterpolateV2GradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp_v2,
                        ops::InterpolateOpV2CUDAKernel<float>,
                        ops::InterpolateOpV2CUDAKernel<double>,
                        ops::InterpolateOpV2CUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(trilinear_interp_v2_grad,
                        ops::InterpolateV2GradOpCUDAKernel<float>,
                        ops::InterpolateV2GradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(linear_interp_v2, ops::InterpolateOpV2CUDAKernel<float>,
                        ops::InterpolateOpV2CUDAKernel<double>,
                        ops::InterpolateOpV2CUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(linear_interp_v2_grad,
                        ops::InterpolateV2GradOpCUDAKernel<float>,
                        ops::InterpolateV2GradOpCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(bicubic_interp_v2,
                        ops::InterpolateOpV2CUDAKernel<float>,
                        ops::InterpolateOpV2CUDAKernel<double>,
                        ops::InterpolateOpV2CUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(bicubic_interp_v2_grad,
                        ops::InterpolateV2GradOpCUDAKernel<float>,
                        ops::InterpolateV2GradOpCUDAKernel<double>);
