// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/operators/trace_op.h"

namespace paddle {
namespace operators {

template <typename T>
struct IdentityFunctor {
  HOSTDEVICE explicit inline IdentityFunctor() {}

  HOSTDEVICE inline T operator()(const T& x) const { return x; }
};

template <typename DeviceContext, typename T>
class TraceCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* input = context.Input<framework::Tensor>("Input");
    auto* out = context.Output<framework::Tensor>("Out");

    const int64_t offset = context.Attr<int>("offset");
    const int64_t dim1 = context.Attr<int>("dim1");
    const int64_t dim2 = context.Attr<int>("dim2");

    T* out_data = out->mutable_data<T>(context.GetPlace());
    const framework::Tensor diag =
        Diagonal<DeviceContext, T>(context, input, offset, dim1, dim2);
    if (diag.numel() > 0) {
      auto stream = context.cuda_device_context().stream();
      std::vector<int> reduce_dims;
      reduce_dims.push_back(out->dims().size());
      TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
          diag, out, reduce_dims, static_cast<T>(0), hipcub::Sum(),
          IdentityFunctor<T>(), stream);
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace platform = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    trace, ops::TraceCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::TraceCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::TraceCUDAKernel<paddle::platform::CUDADeviceContext,
                         platform::float16>,
    ops::TraceCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TraceCUDAKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    trace_grad, ops::TraceGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::TraceGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::TraceGradKernel<paddle::platform::CUDADeviceContext,
                         platform::float16>,
    ops::TraceGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::TraceGradKernel<paddle::platform::CUDADeviceContext, double>);
