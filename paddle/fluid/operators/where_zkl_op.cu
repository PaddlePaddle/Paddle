#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <paddle/fluid/platform/device_context.h>
#include "paddle/fluid/operators/where_zkl_op.h"

namespace plat = paddle::platform;

namespace platform = paddle::platform;

namespace paddle {
namespace operators {

template <class T>
__global__ void WhereZklCUDAKernel(const bool* condition, const T* x,
                                   const T* y, T* out, int64_t N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    if (condition[id])
      out[id] = x[id];
    else
      out[id] = y[id];

    id += blockDim.x * gridDim.x;
  }
}

template <typename T>
class WhereZklKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(context.GetPlace()), true,
                      platform::errors::PreconditionNotMet(
                          "This kernel only runs on GPU device."));

    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();

    auto* condition = context.Input<Tensor>("Condition");
    auto* x = context.Input<Tensor>("X");
    auto* y = context.Input<Tensor>("Y");
    auto* out = context.Output<Tensor>("Out");

    auto* condtion_data = condition->data<bool>();
    auto* x_data = x->data<T>();
    auto* y_data = y->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());

    int n = x->numel();

    int thread_per_block = 256;
    int block_per_grid = (n + thread_per_block - 1) / thread_per_block;

    WhereZklCUDAKernel<T><<<block_per_grid, thread_per_block, 0, stream>>>(
        condtion_data, x_data, y_data, out_data, n);
  }
};

template <class T>
__global__ void WhereZklCUDAGradKernel(const bool* condition, const T* out,
                                       T* dx, T* dy, int64_t N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    dx[id] = condition[id] ? out[id] : 0;
    dy[id] = condition[id] ? 0 : out[id];

    id += blockDim.x * gridDim.x;
  }
}

template <typename T>
class WhereZklGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(context.GetPlace()), true,
                      platform::errors::PreconditionNotMet(
                          "This kernel only runs on GPU device."));

    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();

    auto* condition = context.Input<Tensor>("Condition");
    auto* out = context.Input<Tensor>(framework::GradVarName("Out"));

    auto* dx = context.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = context.Output<Tensor>(framework::GradVarName("Y"));

    auto* condtion_data = condition->data<bool>();
    auto* out_data = out->data<T>();

    auto* dx_data = dx->mutable_data<T>(context.GetPlace());
    auto* dy_data = dy->mutable_data<T>(context.GetPlace());

    int n = out->numel();

    int thread_per_block = 256;
    int block_per_grid = (n + thread_per_block - 1) / thread_per_block;

    WhereZklCUDAGradKernel<T><<<block_per_grid, thread_per_block, 0, stream>>>(
        condtion_data, out_data, dx_data, dy_data, n);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    where_zkl,
    paddle::operators::WhereZklKernel<paddle::platform::CUDADeviceContext,
                                      float>,
    paddle::operators::WhereZklKernel<paddle::platform::CUDADeviceContext,
                                      double>,
    paddle::operators::WhereZklKernel<paddle::platform::CUDADeviceContext, int>,
    paddle::operators::WhereZklKernel<paddle::platform::CUDADeviceContext,
                                      int64_t>);

REGISTER_OP_CUDA_KERNEL(
    where_zkl_grad,
    paddle::operators::WhereZklGradKernel<paddle::platform::CUDADeviceContext,
                                          float>,
    paddle::operators::WhereZklGradKernel<paddle::platform::CUDADeviceContext,
                                          double>,
    paddle::operators::WhereZklGradKernel<paddle::platform::CUDADeviceContext,
                                          int>,
    paddle::operators::WhereZklGradKernel<paddle::platform::CUDADeviceContext,
                                          int64_t>);
