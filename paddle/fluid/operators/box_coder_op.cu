#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserve.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/box_coder_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void EncodeCenterSizeKernel(const T* prior_box_data,
                                       const T* prior_box_var_data,
                                       const T* target_box_data, const int row,
                                       const int col, const int len,
                                       T* output) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < row * col) {
    const int row_idx = idx / col;
    const int col_idx = idx % col;
    T prior_box_width =
        prior_box_data[col_idx * len + 2] - prior_box_data[col_idx * len];
    T prior_box_height =
        prior_box_data[col_idx * len + 3] - prior_box_data[col_idx * len + 1];
    T prior_box_center_x =
        (prior_box_data[col_idx * len + 2] + prior_box_data[col_idx * len]) / 2;
    T prior_box_center_y = (prior_box_data[col_idx * len + 3] +
                            prior_box_data[col_idx * len + 1]) /
                           2;

    T target_box_center_x =
        (target_box_data[row_idx * len + 2] + target_box_data[row_idx * len]) /
        2;
    T target_box_center_y = (target_box_data[row_idx * len + 3] +
                             target_box_data[row_idx * len + 1]) /
                            2;
    T target_box_width =
        target_box_data[row_idx * len + 2] - target_box_data[row_idx * len];
    T target_box_height =
        target_box_data[row_idx * len + 3] - target_box_data[row_idx * len + 1];

    output[idx * len] = (target_box_center_x - prior_box_center_x) /
                        prior_box_width / prior_box_var_data[col_idx * len];
    output[idx * len + 1] = (target_box_center_y - prior_box_center_y) /
                            prior_box_height /
                            prior_box_var_data[col_idx * len + 1];
    output[idx * len + 2] = log(fabs(target_box_width / prior_box_width)) /
                            prior_box_var_data[col_idx * len + 2];
    output[idx * len + 3] = log(fabs(target_box_height / prior_box_height)) /
                            prior_box_var_data[col_idx * len + 3];
  }
}

template <typename T>
__global__ void DecodeCenterSizeKernel(const T* prior_box_data,
                                       const T* prior_box_var_data,
                                       const T* target_box_data, const int row,
                                       const int col, const int len,
                                       T* output) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < row * col) {
    const int row_idx = idx / col;
    const int col_idx = idx % col;
    T prior_box_width =
        prior_box_data[col_idx * len + 2] - prior_box_data[col_idx * len];
    T prior_box_height =
        prior_box_data[col_idx * len + 3] - prior_box_data[col_idx * len + 1];
    T prior_box_center_x =
        (prior_box_data[col_idx * len + 2] + prior_box_data[col_idx * len]) / 2;
    T prior_box_center_y = (prior_box_data[col_idx * len + 3] +
                            prior_box_data[col_idx * len + 1]) /
                           2;

    T target_box_width = exp(prior_box_var_data[col_idx * len + 2] *
                             target_box_data[row_idx * len + 2]) *
                         prior_box_width;
    T target_box_height = exp(prior_box_var_data[col_idx * len + 3] *
                              target_box_data[row_idx * len + 3]) *
                          prior_box_height;
    T target_box_center_x = prior_box_var_data[col_idx * len] *
                                target_box_data[row_idx * len] *
                                prior_box_width +
                            prior_box_center_x;
    T target_box_center_y = prior_box_var_data[col_idx * len + 1] *
                                target_box_data[row_idx * len + 1] *
                                prior_box_height +
                            prior_box_center_y;

    output[idx * len] = target_box_center_x - target_box_width / 2;
    output[idx * len + 1] = target_box_center_y - target_box_height / 2;
    output[idx * len + 2] = target_box_center_x + target_box_width / 2;
    output[idx * len + 3] = target_box_center_y + target_box_height / 2;
  }
}

template <typename T>
class BoxCoderCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto* prior_box = context.Input<framework::Tensor>("PriorBox");
    auto* prior_box_var = context.Input<framework::Tensor>("PriorBoxVar");
    auto* target_box = context.Input<framework::LoDTensor>("TargetBox");
    auto* output_box = context.Output<framework::Tensor>("OutputBox");

    if (target_box->lod().size()) {
      PADDLE_ENFORCE_EQ(target_box->lod().size(), 1,
                        "Only support 1 level of LoD.");
    }
    auto row = target_box->dims()[0];
    auto col = prior_box->dims()[0];
    auto len = prior_box->dims()[1];
    int block = 512;
    int grid = (row * col + block - 1) / block;
    auto& device_ctx = context.cuda_device_context();

    const T* prior_box_data = prior_box->data<T>();
    const T* prior_box_var_data = prior_box_var->data<T>();
    const T* target_box_data = target_box->data<T>();

    output_box->mutable_data<T>({row, col, len}, context.GetPlace());
    T* output = output_box->data<T>();

    auto code_type = GetBoxCodeType(context.Attr<std::string>("code_type"));
    if (code_type == BoxCodeType::kEncodeCenterSize) {
      EncodeCenterSizeKernel<T><<<grid, block, 0, device_ctx.stream()>>>(
          prior_box_data, prior_box_var_data, target_box_data, row, col, len,
          output);
    } else if (code_type == BoxCodeType::kDecodeCenterSize) {
      DecodeCenterSizeKernel<T><<<grid, block, 0, device_ctx.stream()>>>(
          prior_box_data, prior_box_var_data, target_box_data, row, col, len,
          output);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(box_coder, ops::BoxCoderCUDAKernel<float>,
                        ops::BoxCoderCUDAKernel<double>);
