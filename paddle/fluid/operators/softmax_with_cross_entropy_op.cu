#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/softmax_impl.cuh"
#include "paddle/fluid/operators/softmax_with_cross_entropy_op.h"
#include "paddle/fluid/platform/for_range.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/platform/miopen_helper.h"
#else
#include "paddle/fluid/platform/cudnn_helper.h"
#endif

namespace paddle {
namespace operators {

using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
using Tensor = framework::Tensor;

// Wrapper of log function. Use log(float32) for float16
template <typename T>
static __device__ __forceinline__ T Log(T x) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  AccT logx = std::log(static_cast<AccT>(x));
  return math::TolerableValue<T>()(static_cast<T>(logx));
}

// Wrapper of exp function. Use exp(float32) for float16
template <typename T>
static __device__ __forceinline__ T Exp(T x) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  AccT expx = std::exp(static_cast<AccT>(x));
  return math::TolerableValue<T>()(static_cast<T>(expx));
}

// log2(value)
static inline int Log2Ceil(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) ++log2_value;
  return log2_value;
}

enum class SoftmaxMode { kSoftmax, kLogSoftmax, kCrossEntropy };

/*
  Hard label cross entropy.
*/
template <typename T, bool IgnoreIndex>
__global__ void CrossEntropyHardLabel(T* loss, const T* softmax,
                                      const int64_t* labels, const int n,
                                      const int dim, const int d,
                                      const int ignore_idx) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = ids / d;
  int64_t idx_d = ids % d;

  // thread ids compute loss[ids] using softmax[idx]
  if (ids < n * d) {
    int64_t idx = idx_n * dim * d + labels[ids] * d + idx_d;
    if (IgnoreIndex == true) {
      // IgnoreIndex is true
      if (labels[ids] == ignore_idx) {
        loss[ids] = static_cast<T>(0.0);
      } else {
        loss[ids] = -Log(softmax[idx]);
      }
    } else {
      // IgnoreIndex is false
      loss[ids] = -Log(softmax[idx]);
    }
  }
}

/*
  Hard label cross entropy with exp.
  Input: log softmax
  Output: loss and exp(input)
*/
template <typename T, bool IgnoreIndex>
__global__ void CrossEntropyExpHardLabel(T* loss, T* softmax,
                                         const int64_t* labels, const int n,
                                         const int dim, const int d,
                                         const int ignore_idx) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    if (IgnoreIndex == true) {
      // IgnoreIndex is true
      if (idx_dim == labels[ids]) {
        if (labels[ids] == ignore_idx) {
          loss[ids] = static_cast<T>(0.0);
        } else {
          loss[ids] = -softmax[idx];
        }
      }
    } else {
      // IgnoreIndex is false
      if (labels[ids] >= 0 && labels[ids] < dim) {
        if (labels[ids] == idx_dim) {
          loss[ids] = -softmax[idx];
        }
      } else {
        loss[ids] = static_cast<T>(0.0);
      }
    }
    softmax[idx] = Exp(softmax[idx]);
  }
}

/*
  Core function of softmax with cross entropy forward
    - softmax, SoftmaxMode=kSoftmax
    - log softmax, SoftmaxMode=kLogSoftmax
    - softmax with cross entropy hard label, SoftmaxMode=kCrossEntropy
  The computation includes
    - Compute max value: maxvalue_{i} = max_j src_{i,j}
    - Compute sum of exp: s_{i} = sum_{j}{e^{src_{i,j} - maxvalue_{i}}}
    - Compute: softmax_{i,j} = e^{src_{i,j} - maxvalue_{i}} / s_{i}
    - Compute: logsoftmax_{i,j} = src_{i,j} - maxvalue_{i} - log(s_{i})
    - Compute: loss_{i} = -logsoftmax[i,label[i]] (Hard label)
  This computation results from following formula:
    softmax_{i,j} = e^{src_{i,j}} / sum_{j}{e^{src_{i,j}}}
                  = e^{src_{i,j} - maxvalue_{i}}
                    / sum_{j}{e^{src_{i,j} - maxvalue_{i}}}
                  = e^{src_{i,j} - maxvalue_{i}} / s_{i}
    logsoftmax_{i,j} = log(softmax_{i,j})
                     = src_{i,j} - maxvalue_{i} - log(s_{i})
  One warp (32 threads) is used to compute 1 or 2 batch (kBatchSize).
  For reduction max (sum), firstly compute max (sum) to one warp, then use
  shuffle api to compute max (sum) in one warp.
*/
template <typename T, typename VecT, typename AccT, int Log2Elements,
          SoftmaxMode mode, bool IgnoreIndex>
__global__ void WarpSoftmaxForward(T* loss, T* softmax, const T* src,
                                   const int64_t* label, const int batch_size,
                                   const int stride, const int element_count,
                                   const int ignore_index) {
  constexpr int kDimCeil = 1 << Log2Elements;
  constexpr int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  constexpr int kVSize = sizeof(VecT) / sizeof(T);
  constexpr int kIterations = kDimCeil / kWarpSize;
  constexpr int kIterationsV =
      (kIterations >= kVSize) ? (kIterations / kVSize) : 1;
  constexpr int kBatchSize = (kDimCeil <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;

  // max index to read
  int idx_max_v[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; i++) {
    int idx_max = ((i + first_batch) < batch_size) ? element_count : 0;
    idx_max_v[i] = idx_max / kVSize;
  }

  // read data from global memory
  AccT srcdata[kBatchSize][kIterationsV][kVSize];

#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
// read data to srcdata: - KVSize==1, - KVSize>1
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      int src_idx = threadIdx.x + it * kWarpSize;
      if (kVSize == 1) {
        if (src_idx < idx_max_v[i]) {
          srcdata[i][it][0] =
              static_cast<AccT>(src[(first_batch + i) * stride + src_idx]);
        } else {
          srcdata[i][it][0] = -std::numeric_limits<AccT>::infinity();
        }
      } else {
        const VecT* src_v =
            reinterpret_cast<const VecT*>(&src[(first_batch + i) * stride]);
        if (src_idx < idx_max_v[i]) {
          VecT srctmp = src_v[src_idx];
          const T* srcinptr = reinterpret_cast<const T*>(&srctmp);
#pragma unroll
          for (int s = 0; s < kVSize; s++) {
            srcdata[i][it][s] = static_cast<AccT>(srcinptr[s]);
          }
        } else {
#pragma unroll
          for (int s = 0; s < kVSize; s++) {
            srcdata[i][it][s] = -std::numeric_limits<AccT>::infinity();
          }
        }
      }
    }
  }

  // compute max value: maxvalue_{i} = max_j src_{i,j}
  AccT max_value[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    // it = 0
    AccT valmax = srcdata[i][0][0];
#pragma unroll
    for (int s = 1; s < kVSize; ++s) {
      valmax = (valmax > srcdata[i][0][s]) ? valmax : srcdata[i][0][s];
    }
    max_value[i] = valmax;

// it = 1, 2, ...
#pragma unroll
    for (int it = 1; it < kIterationsV; ++it) {
      AccT valmax = srcdata[i][it][0];
#pragma unroll
      for (int s = 1; s < kVSize; ++s) {
        valmax = (valmax > srcdata[i][it][s]) ? valmax : srcdata[i][it][s];
      }
      max_value[i] = (max_value[i] > valmax) ? max_value[i] : valmax;
    }
  }
  WarpReduceMax<AccT, kBatchSize, kWarpSize>(max_value);

  // compute sum: s_{i} = sum_{j}{ exp(src_{i,j} - maxvalue_{i} }
  AccT sum[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    // it = 0
    if (mode == SoftmaxMode::kLogSoftmax ||
        mode == SoftmaxMode::kCrossEntropy) {
      sum[i] = std::exp(srcdata[i][0][0] - max_value[i]);
    } else {
      srcdata[i][0][0] = std::exp(srcdata[i][0][0] - max_value[i]);
      sum[i] = srcdata[i][0][0];
    }
#pragma unroll
    for (int s = 1; s < kVSize; ++s) {
      if (mode == SoftmaxMode::kLogSoftmax ||
          mode == SoftmaxMode::kCrossEntropy) {
        sum[i] += std::exp(srcdata[i][0][s] - max_value[i]);
      } else {
        srcdata[i][0][s] = std::exp(srcdata[i][0][s] - max_value[i]);
        sum[i] += srcdata[i][0][s];
      }
    }

// it = 1, 2, ...
#pragma unroll
    for (int it = 1; it < kIterationsV; ++it) {
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (mode == SoftmaxMode::kLogSoftmax ||
            mode == SoftmaxMode::kCrossEntropy) {
          sum[i] += std::exp(srcdata[i][it][s] - max_value[i]);
        } else {
          srcdata[i][it][s] = std::exp(srcdata[i][it][s] - max_value[i]);
          sum[i] += srcdata[i][it][s];
        }
      }
    }
  }
  WarpReduceSum<AccT, kBatchSize, kWarpSize>(sum);

// write data
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    if (mode == SoftmaxMode::kLogSoftmax ||
        mode == SoftmaxMode::kCrossEntropy) {
      sum[i] = std::log(sum[i]);
    }

#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      int idx = threadIdx.x + it * kWarpSize;
      if (kVSize == 1) {  // kVSize==1
        if (idx < idx_max_v[i]) {
          if (mode == SoftmaxMode::kLogSoftmax) {  // log softmax
            softmax[(first_batch + i) * stride + idx] =
                srcdata[i][it][0] - max_value[i] - sum[i];
            // softmax with cross entropy hard label
          } else if (mode == SoftmaxMode::kCrossEntropy) {
            AccT logsoftmax = srcdata[i][it][0] - max_value[i] - sum[i];
            // softmax
            softmax[(first_batch + i) * stride + idx] = std::exp(logsoftmax);
            // label
            int loss_idx = (threadIdx.x + it * kWarpSize) * kVSize;
            if (IgnoreIndex == true) {
              // IgnoreIndex is true
              if (label[first_batch + i] == loss_idx) {
                if (label[first_batch + i] != ignore_index) {
                  loss[first_batch + i] = -logsoftmax;
                } else {
                  loss[first_batch + i] = static_cast<T>(0.0);
                }
              }
            } else {
              // IgnoreIndex is false
              if (label[first_batch + i] >= 0 &&
                  label[first_batch + i] < element_count) {
                if (label[first_batch + i] == loss_idx) {
                  loss[first_batch + i] = -logsoftmax;
                }
              } else {
                loss[first_batch + i] = static_cast<T>(0.0);
              }
            }
          } else {  // softmax
            softmax[(first_batch + i) * stride + idx] =
                srcdata[i][it][0] / sum[i];
          }
        } else {
          break;
        }
      } else {  // KVSize>1
        VecT* softmax_v =
            reinterpret_cast<VecT*>(&softmax[(first_batch + i) * stride]);
        VecT tmpdata;
        T* tmpptr = reinterpret_cast<T*>(&tmpdata);
#pragma unroll
        for (int s = 0; s < kVSize; ++s) {
          if (mode == SoftmaxMode::kLogSoftmax) {  // log softmax
            tmpptr[s] = srcdata[i][it][s] - max_value[i] - sum[i];
            // softmax with cross entropy hard label
          } else if (mode == SoftmaxMode::kCrossEntropy) {
            AccT logsoftmax = srcdata[i][it][s] - max_value[i] - sum[i];
            // softmax
            tmpptr[s] = std::exp(logsoftmax);
            // label
            int loss_idx = (threadIdx.x + it * kWarpSize) * kVSize + s;
            if (IgnoreIndex == true) {
              // IgnoreIndex is true
              if (label[first_batch + i] == loss_idx &&
                  label[first_batch + i] != ignore_index) {
                loss[first_batch + i] = -logsoftmax;
              }
            } else {
              // IgnoreIndex is false
              if (label[first_batch + i] >= 0 &&
                  label[first_batch + i] < element_count) {
                if (label[first_batch + i] == loss_idx) {
                  loss[first_batch + i] = -logsoftmax;
                }
              } else {
                loss[first_batch + i] = static_cast<T>(0.0);
              }
            }
          } else {  // softmax
            tmpptr[s] = srcdata[i][it][s] / sum[i];
          }
        }
        if (idx < idx_max_v[i]) {
          softmax_v[idx] = tmpdata;
        } else {
          break;
        }
      }
    }
  }
}

#define SOFTMAX_WARP_FORWARD_CASE(Log2Elements, VecT, AccT)           \
  case Log2Elements:                                                  \
    WarpSoftmaxForward<T, VecT, AccT, Log2Elements, mode,             \
                       IgnoreIndex><<<blocks, threads, 0, stream>>>(  \
        loss, softmax, src, label, batch_size, stride, element_count, \
        ignore_index);                                                \
    break;

/*
  Wrapper of softmax with cross entropy forward hard label.
*/
template <typename T, SoftmaxMode mode, bool IgnoreIndex>
void SwitchWarpSoftmaxForward(T* loss, T* softmax, const T* src,
                              const int64_t* label, const int batch_size,
                              const int stride, const int element_count,
                              const int ignore_index, gpuStream_t stream) {
  using AccT = typename details::MPTypeTrait<T>::Type;

  // use 128 threads per block to maximimize gpu utilization
  const int Log2Elements = static_cast<int>(Log2Ceil(element_count));
  const int kDimCeil = 1 << Log2Elements;
  int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  int batches_per_warp = (kDimCeil <= 128) ? 2 : 1;
  constexpr int threads_per_block = 128;
  int warps_per_block = (threads_per_block / kWarpSize);
  int batches_per_block = warps_per_block * batches_per_warp;
  int blocks = (batch_size + batches_per_block - 1) / batches_per_block;
  dim3 threads(kWarpSize, warps_per_block, 1);

  switch (Log2Elements) {
    SOFTMAX_WARP_FORWARD_CASE(0, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(1, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(2, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(3, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(4, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(5, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(6, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(7, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(8, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(9, T, AccT);
    default:
      break;
  }
}

/*
  Wrapper of softmax with cross entropy hard label.
  - SwitchWarpSoftmaxForward for small size
  - cudnn function for large size
*/
template <typename T, bool IgnoreIndex>
static void SoftmaxWithCrossEntropyHardLabel(
    const platform::CUDADeviceContext& ctx, int rank, int axis,
    const T* logits_data, const int64_t* labels_data, T* loss_data,
    T* softmax_data, int N, int dim, int D, const int ignore_index) {
  auto stream = ctx.stream();
  constexpr int max_dim = 320;
  if (D == 1 && dim <= max_dim) {  // small size
    const SoftmaxMode mode = SoftmaxMode::kCrossEntropy;
    SwitchWarpSoftmaxForward<T, mode, IgnoreIndex>(
        loss_data, softmax_data, logits_data, labels_data, N, dim, dim,
        ignore_index, stream);
  } else {
    ScopedTensorDescriptor desc;
    std::vector<int> tensor_dims = {N, dim, D, 1};
    DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
    miopenTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#else
    hipdnnTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#endif

    auto handle = ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
    auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                 : MIOPEN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxForward_V2(
        handle, platform::CudnnDataType<T>::kOne(), descp, logits_data,
        platform::CudnnDataType<T>::kZero(), descp, softmax_data,
        MIOPEN_SOFTMAX_LOG, mode));
#else
    auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                 : HIPDNN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_LOG, mode, platform::CudnnDataType<T>::kOne(),
        descp, logits_data, platform::CudnnDataType<T>::kZero(), descp,
        softmax_data));
#endif
    int threads = 128;
    int blocks = (N * dim * D + threads - 1) / threads;
    // compute cross entropy, input is log softmax
    CrossEntropyExpHardLabel<T, IgnoreIndex><<<blocks, threads, 0, stream>>>(
        loss_data, softmax_data, labels_data, N, dim, D, ignore_index);
  }
}

/*
  Wrapper of softmax with cross entropy grad hard label.
*/
template <typename T>
__global__ void SoftmaxWithCrossEntropyGradHardLabel(
    T* logits_grad, const T* loss_grad, const int64_t* labels, const int64_t n,
    const int64_t dim, const int64_t d, const int ignore_index) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    if (labels[ids] == ignore_index) {
      logits_grad[idx] = static_cast<T>(0.0);
    } else if (labels[ids] == idx_dim) {
      logits_grad[idx] =
          (logits_grad[idx] - static_cast<T>(1.0)) * loss_grad[ids];
    } else {
      logits_grad[idx] *= loss_grad[ids];
    }
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels, const int64_t n,
                                               const int64_t d,
                                               const int64_t remain) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int64_t idx_n = ids / d;
    int64_t idx_remain = ids % remain;
    int64_t idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (logit_grad[ids] - labels[ids]);
  }
}

template <typename T>
__global__ void SoftLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const T* loss_grad,
                                                    const T* labels,
                                                    const int n, const int d,
                                                    const int remain) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int idx_n = ids / d;
    int idx_remain = ids % remain;
    int idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (-labels[ids] / logit_grad[ids]);
  }
}

template <typename T>
__global__ void HardLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const int64_t* labels,
                                                    const int n, const int d,
                                                    const int remain,
                                                    const int ignore_index) {
  CUDA_KERNEL_LOOP(index, n * remain) {
    int idx_n = index / remain;
    int idx_remain = index % remain;
    int tmp = labels[index];
    int idx = idx_n * d + tmp * remain + idx_remain;
    if (ignore_index != tmp) {
      logit_grad[idx] = -static_cast<T>(1.) / logit_grad[idx];
    }
  }
}

template <typename T>
__global__ void ScaleCrossEntropyGradient(T* logit_grad, const T* loss_grad,
                                          const int num, const int d,
                                          const int remain,
                                          const int64_t* labels,
                                          const int ignore_index) {
  CUDA_KERNEL_LOOP(index, num) {
    int idx_n = index / d;
    int idx_remain = index % remain;
    int idx_lbl = idx_n * remain + idx_remain;
    int k = (index % d) / remain;
    if (labels[idx_lbl] == ignore_index || labels[idx_lbl] != k) {
      logit_grad[index] = static_cast<T>(0.);
    } else {
      logit_grad[index] *= loss_grad[idx_lbl];
    }
  }
}

static __device__ __forceinline__ platform::float16 exp_on_device(
    platform::float16 x) {
  return ::Eigen::numext::exp(x);
}
static __device__ __forceinline__ float exp_on_device(float x) {
  return expf(x);
}
static __device__ __forceinline__ double exp_on_device(double x) {
  return exp(x);
}
static __device__ __forceinline__ platform::float16 log_on_device(
    platform::float16 x) {
  return math::TolerableValue<platform::float16>()(::Eigen::numext::log(x));
}
static __device__ __forceinline__ float log_on_device(float x) {
  return math::TolerableValue<float>()(logf(x));
}
static __device__ __forceinline__ double log_on_device(double x) {
  return math::TolerableValue<double>()(log(x));
}

/** In the following codes, 3 CUDA kernels are implemented to calculate softmax
 * and loss **/
/*
  Supposing the x is `logits` and y is `labels`, the equations are as
followings:
  cross\_entropy_i = \sum_{j}[- y_i_j * log({e^{x_i_j}/\sum_{j}e^{x_i_j}})]
        = \sum_{j}[- y_i_j * log({e^{x_i_j - max_i}/\sum_{j}e^{x_i_j-max_i}})]
        = \sum_{j}[-y_i_j * (x_i_j - max_i - log\sum_{j}e^{x_i_j - max_i})]
        = \sum_{j}[-y_i_j * (x_i_j - max_i - logDiffMaxSum_i)]
        = \sum_{j}(-y_i_j * tmp_i_j)
  softmax_i_j = e^{tmp_i_j}
where:
  max_i = \max_{j}{x_i_j}
  logDiffMaxSum_i = log\sum_{j}e^{x_i_j - max_i}
  tmp_i_j = x_i_j - max_i - logDiffMaxSum_i
Therefore, the calculation can be separated into 3 steps:
Step 1: row-wise operation to calculate max_i
Step 2: row-wise operation to calculate logDiffMaxSum_i
Step 3: calculate tmp_i_j, and finally get softmax_i_j and cross\_entropy_i
To save memory, we can share memory among max_i, logDiffMaxSum_i and
cross\_entropy_i.
In this way, the 3 steps should be changed to:
Step 1 (RowReductionForMax): row-wise operation to calculate max_i
Step 2 (RowReductionForDiffMaxSum): calculate immediate result of softmax'_i_j =
x_i_j - max_i, and row-wise operation to calculate logDiffMaxSum_i
Step 3 (RowReductionForSoftmaxAndCrossEntropy): calculate tmp_i_j = softmax'_i_j
- logDiffMaxSum_i, and finally get softmax_i_j and cross\_entropy_i
*/

// There are 3 kinds of reduce algorithms in cub:
// BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY
// BLOCK_REDUCE_RAKING
// BLOCK_REDUCE_WARP_REDUCTIONS (default)
template <typename T, int BlockDim>
using BlockReduce =
    hipcub::BlockReduce<T, BlockDim /*, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS*/>;

template <typename T, int BlockDim>
using BlockReduceTempStorage = typename BlockReduce<T, BlockDim>::TempStorage;

// Make sure that BlockDim <= axis_dim
// This kernel is used to calculate the max element of each row
template <typename T, int BlockDim>
static __global__ void RowReductionForMax(const T* logits_data, T* max_data,
                                          int64_t d, int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits_data view as [n, axis_dim, remain]
  // max_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int64_t remain = d / axis_dim;
  int64_t idx_n = blockIdx.x / remain;
  int64_t idx_remain = blockIdx.x % remain;
  int64_t beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int64_t end_idx = (idx_n + 1) * d;

  int64_t step = BlockDim * remain;
  T cur_max = logits_data[beg_idx];
  beg_idx += step;
  while (beg_idx < end_idx) {
    if (cur_max < logits_data[beg_idx]) {
      cur_max = logits_data[beg_idx];
    }
    beg_idx += step;
  }

  cur_max = BlockReduce<T, BlockDim>(temp_storage).Reduce(cur_max, hipcub::Max());

  if (threadIdx.x == 0) max_data[blockIdx.x] = cur_max;
}

// Make sure that BlockDim <= axis_dim
template <typename T, int BlockDim, bool CalculateLogSoftmax = false>
static __global__ void RowReductionForDiffMaxSum(const T* logits_data,
                                                 T* max_data, T* softmax,
                                                 int64_t d, int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits, softmax data view as [n, axis_dim, remain]
  // max_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int64_t remain = d / axis_dim;
  int64_t idx_n = blockIdx.x / remain;
  int64_t idx_remain = blockIdx.x % remain;
  int64_t beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int64_t end_idx = (idx_n + 1) * d;

  auto block_max = max_data[blockIdx.x];
  int64_t step = BlockDim * remain;

  // In numeric stable mode softmax_with_loss, we calc loss with
  // tmp_i_j = x_i_j - max_i - logDiffMaxSum_i, instead of
  // log(exp(x_i_j - max_i)/DiffMaxSum_i). Therefore, log(0) will not occur.
  // Also we calc softmax_i_j = e^{tmp_i_j}, the maximum and minimum value will
  // be 1.0 and 0.0, represent prob is 1.0 and 0.0.
  // So there is no need to clip on shift_softmax.
  softmax[beg_idx] = logits_data[beg_idx] - block_max;
  T diff_max_sum = exp_on_device(softmax[beg_idx]);
  auto idx = beg_idx + step;
  while (idx < end_idx) {
    softmax[idx] = logits_data[idx] - block_max;
    diff_max_sum += exp_on_device(softmax[idx]);
    idx += step;
  }

  diff_max_sum =
      BlockReduce<T, BlockDim>(temp_storage).Reduce(diff_max_sum, hipcub::Sum());
  if (threadIdx.x == 0) max_data[blockIdx.x] = log_on_device(diff_max_sum);

  if (!CalculateLogSoftmax) return;
  __syncthreads();
  diff_max_sum = max_data[blockIdx.x];
  softmax[beg_idx] -= diff_max_sum;
  beg_idx += step;
  while (beg_idx < end_idx) {
    softmax[beg_idx] -= diff_max_sum;
    beg_idx += step;
  }

  // Note(zhiqiu): since different threads may use max_data[blockIdx.x] to
  // calculate diff_max_sum, __syncthreads() is needed here.
  __syncthreads();
  if (threadIdx.x == 0) max_data[blockIdx.x] = 0;
}

#ifdef __HIPCC__  // @{ HIP Seperate Kernel for RowReductionForDiffMaxSum
// Note(qili93): HIP do not support return in kernel, need to seperate
// RowReductionForDiffMaxSum into two kernels below
template <typename T, int BlockDim>
static __global__ void RowReductionForSum(const T* logits_data, T* max_data,
                                          T* softmax, int64_t d, int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  int64_t remain = d / axis_dim;
  int64_t idx_n = blockIdx.x / remain;
  int64_t idx_remain = blockIdx.x % remain;
  int64_t beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int64_t end_idx = (idx_n + 1) * d;

  auto block_max = max_data[blockIdx.x];
  int64_t step = BlockDim * remain;

  softmax[beg_idx] = logits_data[beg_idx] - block_max;
  T diff_max_sum = exp_on_device(softmax[beg_idx]);
  auto idx = beg_idx + step;
  while (idx < end_idx) {
    softmax[idx] = logits_data[idx] - block_max;
    diff_max_sum += exp_on_device(softmax[idx]);
    idx += step;
  }

  diff_max_sum =
      BlockReduce<T, BlockDim>(temp_storage).Reduce(diff_max_sum, hipcub::Sum());
  if (threadIdx.x == 0) max_data[blockIdx.x] = log_on_device(diff_max_sum);
}

template <typename T, int BlockDim, bool CalculateLogSoftmax = false>
static __global__ void RowReductionForDiff(const T* logits_data, T* max_data,
                                           T* softmax, int d, int axis_dim) {
  int remain = d / axis_dim;
  int idx_n = blockIdx.x / remain;
  int idx_remain = blockIdx.x % remain;
  int beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int end_idx = (idx_n + 1) * d;
  int step = BlockDim * remain;

  T diff_max_sum = max_data[blockIdx.x];
  softmax[beg_idx] -= diff_max_sum;
  beg_idx += step;
  while (beg_idx < end_idx) {
    softmax[beg_idx] -= diff_max_sum;
    beg_idx += step;
  }

  __syncthreads();
  if (threadIdx.x == 0) max_data[blockIdx.x] = 0;
}
#endif  // @} End HIP Seperate Kernel for RowReductionForDiffMaxSum

// Make sure that BlockDim <= axis_dim
template <typename T, int BlockDim>
static __global__ void RowReductionForSoftmaxAndCrossEntropy(
    const T* logits_data, const T* labels_data, T* loss_data, T* softmax,
    int64_t d, int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits, softmax, labels data view as [n, axis_dim, remain]
  // loss_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int64_t remain = d / axis_dim;
  int64_t idx_n = blockIdx.x / remain;
  int64_t idx_remain = blockIdx.x % remain;
  int64_t beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int64_t end_idx = (idx_n + 1) * d;

  // log_diff_max_sum shares memory with loss
  auto block_log_diff_max_sum = loss_data[blockIdx.x];
  auto tmp = softmax[beg_idx] - block_log_diff_max_sum;
  softmax[beg_idx] = exp_on_device(tmp);
  auto loss = -labels_data[beg_idx] * tmp;
  int64_t step = BlockDim * remain;
  beg_idx += step;
  while (beg_idx < end_idx) {
    tmp = softmax[beg_idx] - block_log_diff_max_sum;
    softmax[beg_idx] = exp_on_device(tmp);
    loss -= (labels_data[beg_idx] * tmp);
    beg_idx += step;
  }

  loss = BlockReduce<T, BlockDim>(temp_storage).Reduce(loss, hipcub::Sum());
  if (threadIdx.x == 0) loss_data[blockIdx.x] = loss;
}

// Make sure that BlockDim <= axis_dim
template <typename T, int BlockDim>
static __global__ void RowReductionForCrossEntropy(const T* logits_data,
                                                   const T* labels_data,
                                                   T* loss_data, int d,
                                                   int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits, softmax, labels data view as [n, axis_dim, remain]
  // loss_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int remain = d / axis_dim;
  int idx_n = blockIdx.x / remain;
  int idx_remain = blockIdx.x % remain;
  int beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int end_idx = (idx_n + 1) * d;

  // log_diff_max_sum shares memory with loss
  auto block_log_diff_max_sum = loss_data[blockIdx.x];
  auto tmp = log_on_device(logits_data[beg_idx]);  // when not with softmax,
                                                   // softmax is stored in
                                                   // logits_data
  auto loss = -labels_data[beg_idx] * tmp;
  int step = BlockDim * remain;
  beg_idx += step;
  while (beg_idx < end_idx) {
    tmp = log_on_device(logits_data[beg_idx]);  // when not with softmax,
                                                // softmax is stored in
                                                // logits_data
    loss -= (labels_data[beg_idx] * tmp);
    beg_idx += step;
  }

  loss = BlockReduce<T, BlockDim>(temp_storage).Reduce(loss, hipcub::Sum());
  if (threadIdx.x == 0) loss_data[blockIdx.x] = loss;
}

template <typename T>
static void SoftmaxWithCrossEntropyFusedKernel(
    const T* logits_data, const T* labels_data, T* softmax_data, T* loss_data,
    int64_t n, int64_t d, int axis_dim, gpuStream_t stream) {
#ifdef __HIPCC__
  constexpr int kMaxBlockDim = 256;
#else
  constexpr int kMaxBlockDim = 512;
#endif
  int64_t block_dim = axis_dim >= kMaxBlockDim
                          ? kMaxBlockDim
                          : (1 << static_cast<int>(std::log2(axis_dim)));
  int64_t grid_dim = n * d / axis_dim;
#ifdef __HIPCC__
#define CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)                 \
  case BlockDim:                                                               \
    hipLaunchKernelGGL(HIP_KERNEL_NAME(RowReductionForMax<T, BlockDim>),       \
                       dim3(grid_dim), dim3(BlockDim), 0, stream, logits_data, \
                       loss_data, d, axis_dim);                                \
    hipLaunchKernelGGL(HIP_KERNEL_NAME(RowReductionForSum<T, BlockDim>),       \
                       dim3(grid_dim), dim3(BlockDim), 0, stream, logits_data, \
                       loss_data, softmax_data, d, axis_dim);                  \
    hipLaunchKernelGGL(                                                        \
        HIP_KERNEL_NAME(RowReductionForSoftmaxAndCrossEntropy<T, BlockDim>),   \
        dim3(grid_dim), dim3(BlockDim), 0, stream, logits_data, labels_data,   \
        loss_data, softmax_data, d, axis_dim);                                 \
    break
#else
#define CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)                 \
  case BlockDim:                                                               \
    RowReductionForMax<T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>(        \
        logits_data, loss_data, d, axis_dim);                                  \
    RowReductionForDiffMaxSum<T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>( \
        logits_data, loss_data, softmax_data, d, axis_dim);                    \
    RowReductionForSoftmaxAndCrossEntropy<                                     \
        T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>(                       \
        logits_data, labels_data, loss_data, softmax_data, d, axis_dim);       \
    break
#endif

  switch (block_dim) {
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(512);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(256);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(128);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(64);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(32);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(16);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(8);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(4);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(2);
    default:
      PADDLE_THROW(platform::errors::Unavailable(
          "Block Dimension must be 2^n in softmax_with_cross_entropy_op."));
      break;
  }

#undef CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL
}

// not with softmax
template <typename T>
static void CrossEntropyFusedKernel(const T* logits_data, const T* labels_data,
                                    T* loss_data, int n, int d, int axis_dim,
                                    gpuStream_t stream) {
  constexpr int kMaxBlockDim = 512;
  int block_dim = axis_dim >= kMaxBlockDim
                      ? kMaxBlockDim
                      : (1 << static_cast<int>(std::log2(axis_dim)));
  int grid_dim = n * d / axis_dim;

#define CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)                \
  case BlockDim:                                                              \
    RowReductionForCrossEntropy<T,                                            \
                                BlockDim><<<grid_dim, BlockDim, 0, stream>>>( \
        logits_data, labels_data, loss_data, d, axis_dim);                    \
    break

  switch (block_dim) {
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(512);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(256);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(128);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(64);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(32);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(16);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(8);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(4);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(2);
    default:
      PADDLE_THROW(platform::errors::Unavailable(
          "Block Dimension must be 2^n in softmax_with_cross_entropy_op."));
      break;
  }

#undef CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::Unavailable("softmax_with_cross_entropy operator's "
                                      "CUDA kernel only runs on GPU device."));
    const bool use_softmax = context.Attr<bool>("use_softmax");

    // do not with softmax op, and input is softmax
    if (!use_softmax) {
      const Tensor* softmax = context.Input<Tensor>("Logits");
      const Tensor* labels = context.Input<Tensor>("Label");
      Tensor* softmax_out = context.Output<Tensor>("Softmax");
      Tensor* loss = context.Output<Tensor>("Loss");

      const int rank = softmax->dims().size();
      const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
      const int axis_dim = softmax->dims()[axis];

      const int n = SizeToAxis(axis, softmax->dims());
      const int d = SizeFromAxis(axis, softmax->dims());

      auto* softmax_out_data = softmax_out->mutable_data<T>(context.GetPlace());
      auto* loss_data = loss->mutable_data<T>(context.GetPlace());

      math::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      if (axis_dim == 1) {
        set_constant(context.cuda_device_context(), softmax_out,
                     static_cast<T>(1));
        return;
      }

      auto soft_label = context.Attr<bool>("soft_label");
      auto ignore_index = context.Attr<int>("ignore_index");

      Tensor softmax_2d, labels_2d, loss_2d, softmax_out_2d;
      softmax_2d.ShareDataWith(*softmax).Resize({n, d});
      labels_2d.ShareDataWith(*labels).Resize({n, labels->numel() / n});
      loss_2d.ShareDataWith(*loss).Resize({n, 1});
      softmax_out_2d.ShareDataWith(*softmax_out).Resize({n, d});

      // math::CrossEntropyFunctor support axis is the last
      if (axis == -1) {
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            soft_label, ignore_index, axis_dim);
        return;
      }

      // if axis is not the last, we need a new impliment
      if (soft_label) {
        auto* logits_data = softmax->data<T>();
        auto* labels_data = labels->data<T>();
        CrossEntropyFusedKernel(logits_data, labels_data, loss_data, n, d,
                                axis_dim,
                                context.cuda_device_context().stream());
      } else {  // HardLabel
        auto* logits_data = softmax->data<T>();
        auto* labels_data = labels->data<int64_t>();
        int threads = 128;
        int blocks = (n * d / axis_dim + threads - 1) / threads;
        if (ignore_index >= 0 && ignore_index < axis_dim) {
          CrossEntropyHardLabel<T, true><<<
              blocks, threads, 0, context.cuda_device_context().stream()>>>(
              loss_data, logits_data, labels_data, n, axis_dim, d / axis_dim,
              ignore_index);
        } else {
          CrossEntropyHardLabel<T, false><<<
              blocks, threads, 0, context.cuda_device_context().stream()>>>(
              loss_data, logits_data, labels_data, n, axis_dim, d / axis_dim,
              ignore_index);
        }
      }

      // cause of input is softmax
      // copy to output softmax, directly
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), softmax_out);

      return;
    }

    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Label");
    Tensor* softmax = context.Output<Tensor>("Softmax");
    Tensor* loss = context.Output<Tensor>("Loss");

    const int rank = logits->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logits->dims()[axis];

    const int64_t n = SizeToAxis(axis, logits->dims());
    const int64_t d = SizeFromAxis(axis, logits->dims());

    auto* softmax_data = softmax->mutable_data<T>(context.GetPlace());
    auto* loss_data = loss->mutable_data<T>(context.GetPlace());

    if (axis_dim == 1) {
      math::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), softmax, static_cast<T>(1));
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      return;
    }

    auto soft_label = context.Attr<bool>("soft_label");
    auto ignore_index = context.Attr<int>("ignore_index");

    if (soft_label) {
      auto* logits_data = logits->data<T>();
      auto* labels_data = labels->data<T>();
      SoftmaxWithCrossEntropyFusedKernel(
          logits_data, labels_data, softmax_data, loss_data, n, d, axis_dim,
          context.cuda_device_context().stream());
    } else {
      if (!context.Attr<bool>("numeric_stable_mode")) {
        // CUDNN kernel only suppoer 2-D tensor and perfome softmax on last dim
        Tensor logits_2d, softmax_2d, labels_2d, loss_2d;
        logits_2d.ShareDataWith(*logits).Resize({n, d});
        softmax_2d.ShareDataWith(*softmax).Resize({n, d});
        labels_2d.ShareDataWith(*labels).Resize({n, labels->numel() / n});
        loss_2d.ShareDataWith(*loss).Resize({n, 1});
        math::SoftmaxCUDNNFunctor<T>()(context.cuda_device_context(),
                                       &logits_2d, &softmax_2d);
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            false, ignore_index, axis_dim);
      } else {
        auto* logits_data = logits->data<T>();
        auto* labels_data = labels->data<int64_t>();
        if (ignore_index >= 0 && ignore_index < axis_dim) {
          SoftmaxWithCrossEntropyHardLabel<T, true>(
              context.cuda_device_context(), rank, axis, logits_data,
              labels_data, loss_data, softmax_data, n, axis_dim, d / axis_dim,
              ignore_index);
        } else {
          SoftmaxWithCrossEntropyHardLabel<T, false>(
              context.cuda_device_context(), rank, axis, logits_data,
              labels_data, loss_data, softmax_data, n, axis_dim, d / axis_dim,
              ignore_index);
        }
      }
    }
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::Unavailable("softmax_with_cross_entropy operator's "
                                      "CUDA kernel only runs on GPU device."));
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    const Tensor* softmax = context.Input<Tensor>("Softmax");
    if (logit_grad != softmax) {
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), logit_grad);
    }
    T* logit_grad_data = logit_grad->data<T>();

    const int rank = logit_grad->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logit_grad->dims()[axis];

    const int64_t n = SizeToAxis(axis, logit_grad->dims());
    const int64_t d = SizeFromAxis(axis, logit_grad->dims());
    const int64_t remain = d / axis_dim;

    int block = 512;
    auto stream = context.cuda_device_context().stream();
    auto ignore_index = context.Attr<int>("ignore_index");
    auto use_softmax = context.Attr<bool>("use_softmax");

    // do not with softmax op, and input is softmax
    if (!use_softmax) {
      if (context.Attr<bool>("soft_label")) {
        int grid = (n * d + block - 1) / block;
        const T* label_data = labels->data<T>();
        SoftLabelCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
            logit_grad_data, loss_grad_data, label_data, n, d, remain);
      } else {
        Tensor logits_grad_2d;
        logits_grad_2d.ShareDataWith(*logit_grad).Resize({n, d});
        int grid = (n * remain + block - 1) / block;
        const int64_t* label_data = labels->data<int64_t>();
        HardLabelCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
            logit_grad_data, label_data, n, d, remain, ignore_index);
        int num = n * d;
        grid = (num + block - 1) / block;
        ScaleCrossEntropyGradient<T><<<grid, block, 0, stream>>>(
            logit_grad_data, loss_grad_data, num, d, remain, label_data,
            ignore_index);
      }

      return;
    }

    // with softmax, continue

    if (context.Attr<bool>("soft_label")) {
      int64_t grid = (n * d + block - 1) / block;
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d, remain);
    } else {
      const int64_t* label_data = labels->data<int64_t>();
      int grid = (n * d + block - 1) / block;
      SoftmaxWithCrossEntropyGradHardLabel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d / remain, remain,
          ignore_index);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>);
#else
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<double>);
#endif
