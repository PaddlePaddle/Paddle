#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/operators/softmax_with_cross_entropy_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpudnn/softmax_gpudnn.h"

namespace paddle {
namespace operators {

#define ALIGN_BYTES 16

using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
using Tensor = framework::Tensor;
namespace kps = phi::kps;

// Wrapper of log function. Use log(float32) for float16
template <typename T>
static __device__ __forceinline__ T Log(T x) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  AccT logx = std::log(static_cast<AccT>(x));
  return math::TolerableValue<T>()(static_cast<T>(logx));
}

// Wrapper of exp function. Use exp(float32) for float16
template <typename T>
static __device__ __forceinline__ T Exp(T x) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  AccT expx = std::exp(static_cast<AccT>(x));
  return math::TolerableValue<T>()(static_cast<T>(expx));
}

template <typename Tx, typename Ty = Tx>
struct ExpAddFunctor {
  HOSTDEVICE inline ExpAddFunctor(Tx max) : max(max) {}

  HOSTDEVICE inline Ty operator()(const Tx& sum, const Tx& x) const {
    return static_cast<Ty>(sum + std::exp(x - max));
  }

 private:
  Tx max;
};

// log2(value)
static inline int Log2Ceil(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) ++log2_value;
  return log2_value;
}

enum class SoftmaxMode { kSoftmax, kLogSoftmax, kCrossEntropy };

/*
  Hard label cross entropy.
*/
template <typename T, typename LabelT, bool IgnoreIndex>
__global__ void CrossEntropyHardLabel(T* loss, const T* softmax,
                                      const LabelT* labels, const int n,
                                      const int dim, const int d,
                                      const int ignore_idx) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = ids / d;
  int64_t idx_d = ids % d;

  // thread ids compute loss[ids] using softmax[idx]
  if (ids < n * d) {
    auto lbl = static_cast<int64_t>(labels[ids]);
    if (lbl < 0) {  // label is negative
      loss[ids] = static_cast<T>(0.0);
    } else {  // label is positive of zero
      int64_t idx = idx_n * dim * d + lbl * d + idx_d;
      if (IgnoreIndex == true) {
        // IgnoreIndex is true
        if (lbl == ignore_idx) {
          loss[ids] = static_cast<T>(0.0);
        } else {
          loss[ids] = -Log(softmax[idx]);
        }
      } else {
        // IgnoreIndex is false
        loss[ids] = -Log(softmax[idx]);
      }
    }
  }
}

/*
  Hard label cross entropy with exp.
  Input: log softmax
  Output: loss and exp(input)
*/
template <typename T, typename LabelT, bool IgnoreIndex>
__global__ void CrossEntropyExpHardLabel(T* loss, T* softmax,
                                         const LabelT* labels, const int n,
                                         const int dim, const int d,
                                         const int ignore_idx) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    auto lbl = static_cast<int64_t>(labels[ids]);
    if (IgnoreIndex == true) {
      // IgnoreIndex is true
      if (idx_dim == lbl) {
        if (lbl == ignore_idx) {
          loss[ids] = static_cast<T>(0.0);
        } else {
          loss[ids] = -softmax[idx];
        }
      }
    } else {
      // IgnoreIndex is false
      if (lbl >= 0 && lbl < dim) {
        if (lbl == idx_dim) {
          loss[ids] = -softmax[idx];
        }
      } else {
        loss[ids] = static_cast<T>(0.0);
      }
    }
    softmax[idx] = Exp(softmax[idx]);
  }
}

/*
  Core function of softmax with cross entropy forward
    - softmax, SoftmaxMode=kSoftmax
    - log softmax, SoftmaxMode=kLogSoftmax
    - softmax with cross entropy hard label, SoftmaxMode=kCrossEntropy
  The computation includes
    - Compute max value: maxvalue_{i} = max_j src_{i,j}
    - Compute sum of exp: s_{i} = sum_{j}{e^{src_{i,j} - maxvalue_{i}}}
    - Compute: softmax_{i,j} = e^{src_{i,j} - maxvalue_{i}} / s_{i}
    - Compute: logsoftmax_{i,j} = src_{i,j} - maxvalue_{i} - log(s_{i})
    - Compute: loss_{i} = -logsoftmax[i,label[i]] (Hard label)
  This computation results from following formula:
    softmax_{i,j} = e^{src_{i,j}} / sum_{j}{e^{src_{i,j}}}
                  = e^{src_{i,j} - maxvalue_{i}}
                    / sum_{j}{e^{src_{i,j} - maxvalue_{i}}}
                  = e^{src_{i,j} - maxvalue_{i}} / s_{i}
    logsoftmax_{i,j} = log(softmax_{i,j})
                     = src_{i,j} - maxvalue_{i} - log(s_{i})
  One warp (32 threads) is used to compute 1 or 2 batch (kBatchSize).
  For reduction max (sum), firstly compute max (sum) to one warp, then use
  shuffle api to compute max (sum) in one warp.
*/
template <typename T, typename LabelT, typename VecT, typename AccT,
          int Log2Elements, SoftmaxMode mode, bool IgnoreIndex>
__global__ void WarpSoftmaxForward(T* loss, T* softmax, const T* src,
                                   const LabelT* label, const int batch_size,
                                   const int stride, const int element_count,
                                   const int ignore_index) {
  constexpr int kDimCeil = 1 << Log2Elements;
  constexpr int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  constexpr int kVSize = sizeof(VecT) / sizeof(T);
  constexpr int kIterations = kDimCeil / kWarpSize;
  constexpr int kIterationsV =
      (kIterations >= kVSize) ? (kIterations / kVSize) : 1;
  constexpr int kBatchSize = (kDimCeil <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;

  // max index to read
  int idx_max_v[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; i++) {
    int idx_max = ((i + first_batch) < batch_size) ? element_count : 0;
    idx_max_v[i] = idx_max / kVSize;
  }

  // read data from global memory
  AccT srcdata[kBatchSize][kIterationsV][kVSize];

#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
// read data to srcdata: - KVSize==1, - KVSize>1
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      int src_idx = threadIdx.x + it * kWarpSize;
      if (kVSize == 1) {
        if (src_idx < idx_max_v[i]) {
          srcdata[i][it][0] =
              static_cast<AccT>(src[(first_batch + i) * stride + src_idx]);
        } else {
          srcdata[i][it][0] = -std::numeric_limits<AccT>::infinity();
        }
      } else {
        const VecT* src_v =
            reinterpret_cast<const VecT*>(&src[(first_batch + i) * stride]);
        if (src_idx < idx_max_v[i]) {
          VecT srctmp = src_v[src_idx];
          const T* srcinptr = reinterpret_cast<const T*>(&srctmp);
#pragma unroll
          for (int s = 0; s < kVSize; s++) {
            srcdata[i][it][s] = static_cast<AccT>(srcinptr[s]);
          }
        } else {
#pragma unroll
          for (int s = 0; s < kVSize; s++) {
            srcdata[i][it][s] = -std::numeric_limits<AccT>::infinity();
          }
        }
      }
    }
  }

  // compute max value: maxvalue_{i} = max_j src_{i,j}
  AccT max_value[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    // it = 0
    AccT valmax = srcdata[i][0][0];
#pragma unroll
    for (int s = 1; s < kVSize; ++s) {
      valmax = (valmax > srcdata[i][0][s]) ? valmax : srcdata[i][0][s];
    }
    max_value[i] = valmax;

// it = 1, 2, ...
#pragma unroll
    for (int it = 1; it < kIterationsV; ++it) {
      AccT valmax = srcdata[i][it][0];
#pragma unroll
      for (int s = 1; s < kVSize; ++s) {
        valmax = (valmax > srcdata[i][it][s]) ? valmax : srcdata[i][it][s];
      }
      max_value[i] = (max_value[i] > valmax) ? max_value[i] : valmax;
    }
  }
  phi::WarpReduceMax<AccT, kBatchSize, kWarpSize>(max_value);

  // compute sum: s_{i} = sum_{j}{ exp(src_{i,j} - maxvalue_{i} }
  AccT sum[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    // it = 0
    if (mode == SoftmaxMode::kLogSoftmax ||
        mode == SoftmaxMode::kCrossEntropy) {
      sum[i] = std::exp(srcdata[i][0][0] - max_value[i]);
    } else {
      srcdata[i][0][0] = std::exp(srcdata[i][0][0] - max_value[i]);
      sum[i] = srcdata[i][0][0];
    }
#pragma unroll
    for (int s = 1; s < kVSize; ++s) {
      if (mode == SoftmaxMode::kLogSoftmax ||
          mode == SoftmaxMode::kCrossEntropy) {
        sum[i] += std::exp(srcdata[i][0][s] - max_value[i]);
      } else {
        srcdata[i][0][s] = std::exp(srcdata[i][0][s] - max_value[i]);
        sum[i] += srcdata[i][0][s];
      }
    }

// it = 1, 2, ...
#pragma unroll
    for (int it = 1; it < kIterationsV; ++it) {
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (mode == SoftmaxMode::kLogSoftmax ||
            mode == SoftmaxMode::kCrossEntropy) {
          sum[i] += std::exp(srcdata[i][it][s] - max_value[i]);
        } else {
          srcdata[i][it][s] = std::exp(srcdata[i][it][s] - max_value[i]);
          sum[i] += srcdata[i][it][s];
        }
      }
    }
  }
  phi::WarpReduceSum<AccT, kBatchSize, kWarpSize>(sum);

// write data
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    if (mode == SoftmaxMode::kLogSoftmax ||
        mode == SoftmaxMode::kCrossEntropy) {
      sum[i] = std::log(sum[i]);
    }

#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      int idx = threadIdx.x + it * kWarpSize;
      if (kVSize == 1) {  // kVSize==1
        if (idx < idx_max_v[i]) {
          if (mode == SoftmaxMode::kLogSoftmax) {  // log softmax
            softmax[(first_batch + i) * stride + idx] =
                srcdata[i][it][0] - max_value[i] - sum[i];
            // softmax with cross entropy hard label
          } else if (mode == SoftmaxMode::kCrossEntropy) {
            AccT logsoftmax = srcdata[i][it][0] - max_value[i] - sum[i];
            // softmax
            softmax[(first_batch + i) * stride + idx] = std::exp(logsoftmax);
            // label
            int loss_idx = (threadIdx.x + it * kWarpSize) * kVSize;
            auto lbl = static_cast<int64_t>(label[first_batch + i]);
            if (IgnoreIndex == true) {
              // IgnoreIndex is true
              if (lbl == loss_idx) {
                if (lbl != ignore_index) {
                  loss[first_batch + i] = -logsoftmax;
                } else {
                  loss[first_batch + i] = static_cast<T>(0.0);
                }
              }
            } else {
              // IgnoreIndex is false
              if (lbl >= 0 && lbl < element_count) {
                if (lbl == loss_idx) {
                  loss[first_batch + i] = -logsoftmax;
                }
              } else {
                loss[first_batch + i] = static_cast<T>(0.0);
              }
            }
          } else {  // softmax
            softmax[(first_batch + i) * stride + idx] =
                srcdata[i][it][0] / sum[i];
          }
        } else {
          break;
        }
      } else {  // KVSize>1
        VecT* softmax_v =
            reinterpret_cast<VecT*>(&softmax[(first_batch + i) * stride]);
        VecT tmpdata;
        T* tmpptr = reinterpret_cast<T*>(&tmpdata);
#pragma unroll
        for (int s = 0; s < kVSize; ++s) {
          if (mode == SoftmaxMode::kLogSoftmax) {  // log softmax
            tmpptr[s] = srcdata[i][it][s] - max_value[i] - sum[i];
            // softmax with cross entropy hard label
          } else if (mode == SoftmaxMode::kCrossEntropy) {
            AccT logsoftmax = srcdata[i][it][s] - max_value[i] - sum[i];
            // softmax
            tmpptr[s] = std::exp(logsoftmax);
            // label
            int loss_idx = (threadIdx.x + it * kWarpSize) * kVSize + s;
            auto lbl = static_cast<int64_t>(label[first_batch + i]);
            if (IgnoreIndex == true) {
              // IgnoreIndex is true
              if (lbl == loss_idx && lbl != ignore_index) {
                loss[first_batch + i] = -logsoftmax;
              }
            } else {
              // IgnoreIndex is false
              if (lbl >= 0 && lbl < element_count) {
                if (lbl == loss_idx) {
                  loss[first_batch + i] = -logsoftmax;
                }
              } else {
                loss[first_batch + i] = static_cast<T>(0.0);
              }
            }
          } else {  // softmax
            tmpptr[s] = srcdata[i][it][s] / sum[i];
          }
        }
        if (idx < idx_max_v[i]) {
          softmax_v[idx] = tmpdata;
        } else {
          break;
        }
      }
    }
  }
}

#define SOFTMAX_WARP_FORWARD_CASE(Log2Elements, LabelT, VecT, AccT)   \
  case Log2Elements:                                                  \
    WarpSoftmaxForward<T, LabelT, VecT, AccT, Log2Elements, mode,     \
                       IgnoreIndex><<<blocks, threads, 0, stream>>>(  \
        loss, softmax, src, label, batch_size, stride, element_count, \
        ignore_index);                                                \
    break;

/*
  Wrapper of softmax with cross entropy forward hard label.
*/
template <typename T, typename LabelT, SoftmaxMode mode, bool IgnoreIndex>
void SwitchWarpSoftmaxForward(T* loss, T* softmax, const T* src,
                              const LabelT* label, const int batch_size,
                              const int stride, const int element_count,
                              const int ignore_index, gpuStream_t stream) {
  using AccT = typename details::MPTypeTrait<T>::Type;

  // use 128 threads per block to maximimize gpu utilization
  const int log2_elements = static_cast<int>(Log2Ceil(element_count));
  const int kDimCeil = 1 << log2_elements;
  int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  int batches_per_warp = (kDimCeil <= 128) ? 2 : 1;
  constexpr int threads_per_block = 128;
  int warps_per_block = (threads_per_block / kWarpSize);
  int batches_per_block = warps_per_block * batches_per_warp;
  int blocks = (batch_size + batches_per_block - 1) / batches_per_block;
  dim3 threads(kWarpSize, warps_per_block, 1);

  switch (log2_elements) {
    SOFTMAX_WARP_FORWARD_CASE(0, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(1, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(2, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(3, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(4, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(5, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(6, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(7, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(8, LabelT, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(9, LabelT, T, AccT);
    default:
      break;
  }
}

template <typename T, bool IgnoreIndex>
__device__ __forceinline__ void ComputeLoss(T* loss, const T loss_value,
                                            const int label_id,
                                            const int64_t label_value,
                                            const int tid, const int vec_size,
                                            const int offset,
                                            const int ignore_index) {
  int loss_id = vec_size * tid + offset;
  if (IgnoreIndex) {
    if (label_value == loss_id) {
      if (label_value == ignore_index) {
        loss[label_id] = static_cast<T>(0.0f);
      } else {
        loss[label_id] = loss_value;
      }
    }
  } else {
    if (label_value == loss_id) {
      loss[label_id] = loss_value;
    }
  }
}

template <typename T, typename AccT, int VecSize, class ReduceFunctor>
__device__ __forceinline__ AccT ThreadReduce(const T* input, int size,
                                             const int offset, AccT init,
                                             ReduceFunctor reducer) {
  using VecT = kps::details::VectorType<T, VecSize>;
  int tid = threadIdx.x;
  AccT val = init;

  if (offset > 0) {
    input -= offset;
    size += offset;
    if (tid >= offset) {
      val = reducer(val, input[tid]);
    }
    size -= blockDim.x;
    input += blockDim.x;
  }
  int remain = size % (VecSize * blockDim.x);

  T ins[VecSize];
  VecT* ins_vec = reinterpret_cast<VecT*>(&ins);

  // vector part
  for (; VecSize * tid < (size - remain); tid += blockDim.x) {
    *ins_vec = reinterpret_cast<const VecT*>(input)[tid];

#pragma unroll
    for (int i = 0; i < VecSize; ++i) {
      val = reducer(val, ins[i]);
    }
  }

  // scalar part
  tid = size - remain + threadIdx.x;
  for (; tid < size; tid += blockDim.x) {
    val = reducer(val, input[tid]);
  }
  return val;
}

template <typename T, typename AccT, typename LabelT, int VecSize,
          bool IgnoreIndex>
__device__ __forceinline__ void VectorizedSoftmaxForwardImpl(
    T* loss, T* softmax, const T* logits, const LabelT* label, int size,
    const int offset, const phi::LogSoftmaxForwardFunctor<AccT>& func,
    const int ignore_index) {
  using VecT = kps::details::VectorType<T, VecSize>;
  int tid = threadIdx.x;
  int label_id = blockIdx.x;
  auto label_value = static_cast<int64_t>(label[label_id]);
  const bool label_valid = label_value >= 0 && label_value < size;
  int loss_id_offset = 0;

  if (offset > 0) {
    logits -= offset;
    softmax -= offset;
    size += offset;
    loss_id_offset -= offset;
    if (tid >= offset) {
      AccT log_softmax = func(static_cast<AccT>(logits[tid]));
      softmax[tid] = static_cast<T>(std::exp(log_softmax));
      // loss
      if (label_valid) {
        ComputeLoss<T, IgnoreIndex>(loss, static_cast<T>(-log_softmax),
                                    label_id, label_value, tid, 1,
                                    loss_id_offset, ignore_index);
      }
    }
    size -= blockDim.x;
    logits += blockDim.x;
    softmax += blockDim.x;
    loss_id_offset += blockDim.x;
  }
  int remain = size % (VecSize * blockDim.x);

  T ins[VecSize];
  T outs[VecSize];
  VecT* ins_vec = reinterpret_cast<VecT*>(&ins);
  VecT* outs_vec = reinterpret_cast<VecT*>(&outs);

  // vector part
  for (; VecSize * tid < (size - remain); tid += blockDim.x) {
    // read
    *ins_vec = reinterpret_cast<const VecT*>(logits)[tid];

#pragma unroll
    // compute
    for (int i = 0; i < VecSize; ++i) {
      AccT log_softmax = func(static_cast<AccT>(ins[i]));
      outs[i] = static_cast<T>(std::exp(log_softmax));

      // loss
      if (label_valid) {
        ComputeLoss<T, IgnoreIndex>(loss, static_cast<T>(-log_softmax),
                                    label_id, label_value, tid, VecSize,
                                    loss_id_offset + i, ignore_index);
      }
    }

    // write
    reinterpret_cast<VecT*>(softmax)[tid] = *outs_vec;
  }

  // scalar part
  tid = size - remain + threadIdx.x;
  for (; tid < size; tid += blockDim.x) {
    AccT log_softmax = func(static_cast<AccT>(logits[tid]));
    softmax[tid] = static_cast<T>(std::exp(log_softmax));

    // loss
    if (label_valid) {
      ComputeLoss<T, IgnoreIndex>(loss, static_cast<T>(-log_softmax), label_id,
                                  label_value, tid, 1, loss_id_offset,
                                  ignore_index);
    }
  }

  // invalid label, write once
  if (!label_valid && threadIdx.x == 0) {
    loss[label_id] = static_cast<T>(0.0f);
  }
}

template <typename T, typename AccT, typename LabelT, int VecSize,
          bool IgnoreIndex>
__device__ __forceinline__ void ScalarSoftmaxForwardImpl(
    T* loss, T* softmax, const T* logits, const LabelT* label, const int size,
    const phi::LogSoftmaxForwardFunctor<AccT>& func, const int ignore_index) {
  int tid = threadIdx.x;
  int remain = size % (VecSize * blockDim.x);
  int label_id = blockIdx.x;
  auto label_value = static_cast<int64_t>(label[label_id]);
  const bool label_valid = label_value >= 0 && label_value < size;

  // main part
  for (; tid < (size - remain); tid += VecSize * blockDim.x) {
    T ins[VecSize];

#pragma unroll
    for (int i = 0; i < VecSize; ++i) {
      ins[i] = logits[tid + i * blockDim.x];
    }
#pragma unroll
    for (int i = 0; i < VecSize; ++i) {
      AccT log_softmax = func(static_cast<AccT>(ins[i]));
      softmax[tid + i * blockDim.x] = static_cast<T>(std::exp(log_softmax));
      // loss
      if (label_valid) {
        ComputeLoss<T, IgnoreIndex>(loss, static_cast<T>(-log_softmax),
                                    label_id, label_value, tid, VecSize, i,
                                    ignore_index);
      }
    }
  }

  // tail part
  for (; tid < size; tid += blockDim.x) {
    AccT log_softmax = func(static_cast<AccT>(logits[tid]));
    softmax[tid] = static_cast<T>(std::exp(log_softmax));
    // loss
    if (label_valid) {
      ComputeLoss<T, IgnoreIndex>(loss, static_cast<T>(-log_softmax), label_id,
                                  label_value, tid, 1, 0, ignore_index);
    }
  }

  // invalid label, write once
  if (!label_valid && threadIdx.x == 0) {
    loss[label_id] = static_cast<T>(0.0f);
  }
}

template <typename T, typename AccT, typename LabelT, int VecSize,
          bool IgnoreIndex>
__global__ void VectorizedSoftmaxForward(T* loss, T* softmax, const T* logits,
                                         const LabelT* label,
                                         const int high_dim, const int mid_dim,
                                         const int ignore_index) {
  using VecT = kps::details::VectorType<T, VecSize>;

  // each block deal with one batch
  logits += blockIdx.x * mid_dim;
  softmax += blockIdx.x * mid_dim;

  const int input_offset = ((uint64_t)logits) % ALIGN_BYTES / sizeof(T);
  const int output_offset = ((uint64_t)softmax) % ALIGN_BYTES / sizeof(T);

  // 1. reduce max
  AccT max = ThreadReduce<T, AccT, VecSize, kps::MaxFunctor<AccT>>(
      logits, mid_dim, input_offset, -std::numeric_limits<AccT>::infinity(),
      kps::MaxFunctor<AccT>());
  max = kps::details::BlockXReduce<AccT, kps::MaxFunctor<AccT>>(
      max, kps::MaxFunctor<AccT>());

  // 2. reduce sum
  AccT sum = ThreadReduce<T, AccT, VecSize, ExpAddFunctor<AccT>>(
      logits, mid_dim, input_offset, static_cast<AccT>(0),
      ExpAddFunctor<AccT>(max));
  sum = kps::details::BlockXReduce<AccT, kps::AddFunctor<AccT>>(
      sum, kps::AddFunctor<AccT>());

  // 3. softmax
  phi::LogSoftmaxForwardFunctor<AccT> func(max, sum);
  if (input_offset == output_offset) {
    VectorizedSoftmaxForwardImpl<T, AccT, LabelT, VecSize, IgnoreIndex>(
        loss, softmax, logits, label, mid_dim, input_offset, func,
        ignore_index);
  } else {
    ScalarSoftmaxForwardImpl<T, AccT, LabelT, VecSize, IgnoreIndex>(
        loss, softmax, logits, label, mid_dim, func, ignore_index);
  }
}

template <typename T, typename LabelT, bool IgnoreIndex>
void LaunchVectorizedSoftmaxForward(T* loss, T* softmax, const T* logits,
                                    const LabelT* label, const int high_dim,
                                    const int mid_dim, const int ignore_index,
                                    gpuStream_t stream) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  constexpr int vec_size = sizeof(float4) / sizeof(T);
  const int max_num_threads = 1024;
  int max_block_size = std::min(mid_dim / vec_size, max_num_threads);
  if (vec_size > 1) {
    max_block_size /= 2;
  }

  int block_size = 1;
  while (block_size < max_block_size) {
    block_size *= 2;
  }
  block_size = std::max(block_size, kps::details::kWarpSize);
  dim3 grids(high_dim);
  dim3 blocks(block_size);
  VectorizedSoftmaxForward<T, AccT, LabelT, vec_size,
                           IgnoreIndex><<<grids, blocks, 0, stream>>>(
      loss, softmax, logits, label, high_dim, mid_dim, ignore_index);
}

/*
  Wrapper of softmax with cross entropy hard label.
  - SwitchWarpSoftmaxForward for small size when axis == -1
  - LaunchVectorizedSoftmaxForward for large size when axis == -1
  - cudnn function for axis != -1
*/
template <typename T, typename LabelT, bool IgnoreIndex>
static void SoftmaxWithCrossEntropyHardLabel(
    const platform::CUDADeviceContext& ctx, int rank, int axis,
    const T* logits_data, const LabelT* labels_data, T* loss_data,
    T* softmax_data, int N, int dim, int D, const int ignore_index) {
  auto stream = ctx.stream();
  constexpr int max_dim = 320;
  if (D == 1) {
    if (dim <= max_dim) {  // small size
      const SoftmaxMode mode = SoftmaxMode::kCrossEntropy;
      SwitchWarpSoftmaxForward<T, LabelT, mode, IgnoreIndex>(
          loss_data, softmax_data, logits_data, labels_data, N, dim, dim,
          ignore_index, stream);
    } else {  // large size
      LaunchVectorizedSoftmaxForward<T, LabelT, IgnoreIndex>(
          loss_data, softmax_data, logits_data, labels_data, N, dim,
          ignore_index, stream);
    }
  } else {
    ScopedTensorDescriptor desc;
    std::vector<int> tensor_dims = {N, dim, D, 1};
    DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
    miopenTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#else
    hipdnnTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#endif

    auto handle = ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
    auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                 : MIOPEN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::miopenSoftmaxForward_V2(
        handle, platform::CudnnDataType<T>::kOne(), descp, logits_data,
        platform::CudnnDataType<T>::kZero(), descp, softmax_data,
        MIOPEN_SOFTMAX_LOG, mode));
#else
    auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                 : HIPDNN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_LOG, mode, platform::CudnnDataType<T>::kOne(),
        descp, logits_data, platform::CudnnDataType<T>::kZero(), descp,
        softmax_data));
#endif
    int threads = 128;
    int blocks = (N * dim * D + threads - 1) / threads;
    // compute cross entropy, input is log softmax
    CrossEntropyExpHardLabel<T, LabelT,
                             IgnoreIndex><<<blocks, threads, 0, stream>>>(
        loss_data, softmax_data, labels_data, N, dim, D, ignore_index);
  }
}

/*
  Wrapper of softmax with cross entropy grad hard label.
*/
template <typename T, typename LabelT>
__global__ void SoftmaxWithCrossEntropyGradHardLabel(
    T* logits_grad, const T* loss_grad, const LabelT* labels, const int64_t n,
    const int64_t dim, const int64_t d, const int ignore_index) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    auto lbl = static_cast<int64_t>(labels[ids]);
    if (lbl == ignore_index) {
      logits_grad[idx] = static_cast<T>(0.0);
    } else if (lbl == idx_dim) {
      logits_grad[idx] =
          (logits_grad[idx] - static_cast<T>(1.0)) * loss_grad[ids];
    } else {
      logits_grad[idx] *= loss_grad[ids];
    }
  }
}

/*
  Cross entropy soft label with dynamic size on axis (log2_elements is
  varibale).
  - if the input is softmax，compute loss with softmax
  - if the input is log_softmax, compute loss with log_softmax and update
  softmax
*/
template <typename T, typename VecT, bool InLogMode = false>
__global__ void CrossEntropySoftLabel(T* loss, T* softmaxwrt, const T* softmax,
                                      const T* labels, const int n,
                                      const int dim, const int d,
                                      int log2_elements) {
  const int kDimCeil = 1 << log2_elements;
  const int kVSize = sizeof(VecT) / sizeof(T);

#ifdef __HIPCC__
  const int kThreadPerBlock = 256;
#else
  const int kThreadPerBlock = 512;
#endif
  const int kBatchPerBlock = 1;
  const int kWarpSize = 32;  // (dim < 32) ? dim : 32;
  const int kBatchSize = 1;
  const int kThreadPerBatch = kThreadPerBlock / kBatchPerBlock;
  const int kWarpPerBatch = kThreadPerBatch / kWarpSize;

  const int kIterations = (dim + kThreadPerBatch - 1) / kThreadPerBatch;
  const int kIterationsV = (kIterations >= kVSize) ? (kIterations / kVSize) : 1;

  const int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;

  T sum[kBatchSize]{static_cast<T>(0.0)};
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    int ids = first_batch + i;
    if (ids >= n * d) break;
    int idx_n = ids / d;
    int idx_d = ids % d;
#pragma unroll
    for (int it = 0; it < kIterations; ++it) {
      int idx_dim = it * kThreadPerBatch + threadIdx.x;
      int idx = idx_n * dim * d + idx_dim * d + idx_d;

      if (idx_n < n && idx_dim < dim) {
        VecT softmaxdata;
        if (InLogMode) {
          softmaxdata = reinterpret_cast<VecT*>(&softmaxwrt[idx])[0];
        } else {
          softmaxdata = reinterpret_cast<const VecT*>(&softmax[idx])[0];
        }
        VecT labelsdata = reinterpret_cast<const VecT*>(&labels[idx])[0];
        T* softmaxptr = reinterpret_cast<T*>(&softmaxdata);
        T* labelsptr = reinterpret_cast<T*>(&labelsdata);
#pragma unroll
        for (int s = 0; s < kVSize; s++) {
          if (InLogMode) {
            sum[i] -= softmaxptr[s] * labelsptr[s];
            softmaxptr[s] = Exp(softmaxptr[s]);
          } else {
            sum[i] -= Log(softmaxptr[s]) * labelsptr[s];
          }
        }
        if (InLogMode) {
          reinterpret_cast<VecT*>(&softmaxwrt[idx])[0] = softmaxdata;
        }
      }
    }
  }
  phi::WarpReduceSum<T, kBatchSize, kWarpSize>(sum);
  __syncthreads();

  __shared__ T sumshare[kWarpPerBatch][kBatchPerBlock][kBatchSize];
  if (threadIdx.x % kWarpSize == 0) {
#pragma unroll
    for (int i = 0; i < kBatchSize; i++) {
      sumshare[threadIdx.x / kWarpSize][threadIdx.y][i] = sum[i];
    }
  }
  __syncthreads();

  // write
  if (threadIdx.x == 0) {
    for (int i = 0; i < kBatchSize; i++) {
      int ids = first_batch + i;
      if (ids < n * d) {
        loss[ids] = sumshare[0][threadIdx.y][i];
        for (int s = 1; s < kWarpPerBatch; s++) {
          loss[ids] += sumshare[s][threadIdx.y][i];
        }
      }
    }
  }
}

/*
Core function of softmax with cross entropy forward soft label.
The computation includes
  - Compute maximum of batch: maxvalue_{i} = max_j src_{i,j}
  - Compute sum of exp batch: s_{i} = sum_{j}{ exp(src_{i,j} - maxvalue_{i} }
  - Compute: sum of - sum_{j}{ label_{i,j} * (src_{i,j} - maxvalue_{i} -
log(sum[i]))}
One warp (32 threads) is used to compute 1 or 2 batch (kBatchSize).
For reduction max (sum), firstly compute max (sum) to one warp, then use shuffle
api to compute max (sum) in one warp.
*/
template <typename T, typename VecT, typename AccT, int Log2Elements>
__global__ void WarpSoftmaxForwardSoftLabel(T* loss, T* softmax, const T* src,
                                            const T* label,
                                            const int batch_size,
                                            const int stride,
                                            const int element_count) {
  const bool LogMode = true;

  constexpr int kDimCeil = 1 << Log2Elements;
  constexpr int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  constexpr int kVSize = sizeof(VecT) / sizeof(T);
  constexpr int kIterations = kDimCeil / kWarpSize;
  constexpr int kIterationsV =
      (kIterations >= kVSize) ? (kIterations / kVSize) : 1;
  constexpr int kBatchSize = (kDimCeil <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;
  int local_batches = batch_size - first_batch;
  if (local_batches > kBatchSize) {
    local_batches = kBatchSize;
  }

  // read data from global memory
  VecT srcdata[kBatchSize][kIterationsV];
  VecT labeldata[kBatchSize][kIterationsV];

  for (int i = 0; i < kBatchSize; ++i) {
    const VecT* src_v =
        reinterpret_cast<const VecT*>(&src[(first_batch + i) * stride]);
    const VecT* label_v =
        reinterpret_cast<const VecT*>(&label[(first_batch + i) * stride]);

    // max index to read
    int idx_max = (i < local_batches) ? element_count : 0;
    int idx_max_v = idx_max / kVSize;

    // read data
    for (int it = 0; it < kIterationsV; ++it) {
      int src_idx = threadIdx.x + it * kWarpSize;
      if (src_idx < idx_max_v) {
        srcdata[i][it] = src_v[src_idx];
        labeldata[i][it] = label_v[src_idx];
      } else {
#pragma unroll
        for (int s = 0; s < kVSize; s++) {
          reinterpret_cast<T*>(&srcdata[i][it])[s] =
              -std::numeric_limits<AccT>::max();
          reinterpret_cast<T*>(&labeldata[i][it])[s] = 0.0;
        }
      }
    }
  }

  // compute max value
  AccT max_value[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    max_value[i] = -std::numeric_limits<AccT>::infinity();
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      T* srcptr_v = reinterpret_cast<T*>(&srcdata[i][it]);
      T valmax = srcptr_v[0];
#pragma unroll
      for (int s = 1; s < kVSize; ++s) {
        valmax = (valmax > srcptr_v[s]) ? valmax : srcptr_v[s];
      }
      max_value[i] = (max_value[i] > static_cast<AccT>(valmax))
                         ? max_value[i]
                         : static_cast<AccT>(valmax);
    }
  }
  phi::WarpReduceMax<AccT, kBatchSize, kWarpSize>(max_value);

  // compute sum
  AccT sum[kBatchSize]{0.0};
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      T* srcptr_v = reinterpret_cast<T*>(&srcdata[i][it]);
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (LogMode) {
          sum[i] += std::exp(static_cast<AccT>(srcptr_v[s]) - max_value[i]);
        } else {
          srcptr_v[s] = std::exp(static_cast<AccT>(srcptr_v[s]) - max_value[i]);
          sum[i] += static_cast<AccT>(srcptr_v[s]);
        }
      }
    }
  }
  phi::WarpReduceSum<AccT, kBatchSize, kWarpSize>(sum);

  // log_softmax and loss
  AccT sumloss[kBatchSize]{0.0};
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    if (i >= local_batches) break;

    VecT* softmax_v =
        reinterpret_cast<VecT*>(&softmax[(first_batch + i) * stride]);

    // max index to write
    int idx_max = (i < local_batches) ? element_count : 0;
    int idx_max_v = idx_max / kVSize;

    if (LogMode) {
      sum[i] = std::log(sum[i]);
    }
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      T* srcvp = reinterpret_cast<T*>(&srcdata[i][it]);
      T* labelvp = reinterpret_cast<T*>(&labeldata[i][it]);
      VecT tmpv;
      T* tmpvp = reinterpret_cast<T*>(&tmpv);
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (LogMode) {
          AccT logsoftmax = static_cast<AccT>(srcvp[s]) - max_value[i] - sum[i];
          sumloss[i] -= logsoftmax * static_cast<AccT>(labelvp[s]);
          tmpvp[s] = std::exp(logsoftmax);
        } else {
          tmpvp[s] = static_cast<AccT>(srcvp[s]) / sum[i];
        }
      }

      int idx = threadIdx.x + it * kWarpSize;
      if (idx < idx_max_v) {
        softmax_v[idx] = tmpv;
      }
    }
  }

  // loss
  phi::WarpReduceSum<AccT, kBatchSize, kWarpSize>(sumloss);

  for (int i = 0; i < kBatchSize; i++) {
    if (i >= local_batches) break;
    loss[first_batch + i] = sumloss[i];
  }
}

#define SOFTMAX_WARP_FORWARD_SOFT_CASE(Log2Elements, VecT, AccT)               \
  case Log2Elements:                                                           \
    WarpSoftmaxForwardSoftLabel<T, VecT, AccT,                                 \
                                Log2Elements><<<blocks, threads, 0, stream>>>( \
        loss, softmax, src, label, batch_size, stride, element_count);         \
    break;

/*
  Wrapper of softmax with cross entropy forward soft label.
*/
template <typename T>
void SwitchWarpSoftmaxForwardSoftLabel(const int blocks, const dim3 threads,
                                       gpuStream_t stream, T* loss, T* softmax,
                                       const T* src, const T* label,
                                       const int batch_size, const int stride,
                                       const int element_count,
                                       const int log2_elements) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  switch (log2_elements) {
    SOFTMAX_WARP_FORWARD_SOFT_CASE(0, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(1, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(2, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(3, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(4, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(5, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(6, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(7, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(8, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(9, T, AccT);
    default:
      break;
  }
}

template <typename T>
static void SoftmaxWithCrossEntropySoftLabel(
    const platform::CUDADeviceContext& ctx, const int rank, const int axis,
    const T* logits_data, const T* labels_data, T* softmax_data, T* loss_data,
    int N, int dim, int D) {
#ifdef __HIPCC__
  constexpr int kMaxBlockDim = 256;
#else
  constexpr int kMaxBlockDim = 512;
#endif
  int64_t block_dim = dim >= kMaxBlockDim
                          ? kMaxBlockDim
                          : (1 << static_cast<int>(std::log2(dim)));

  int64_t grid_dim = N * D;
  constexpr int max_dim = 320;

  const int kDimLog2 = static_cast<int>(Log2Ceil(dim));
  const int kDimCeil = 1 << kDimLog2;
  auto stream = ctx.stream();

  if (D == 1 && dim <= max_dim) {
    int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
    int batches_per_warp = (kDimCeil <= 128) ? 2 : 1;

    // use 128 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 128;
    int warps_per_block = (threads_per_block / kWarpSize);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = (N + batches_per_block - 1) / batches_per_block;
    dim3 threads(kWarpSize, warps_per_block, 1);

    SwitchWarpSoftmaxForwardSoftLabel<T>(blocks, threads, stream, loss_data,
                                         softmax_data, logits_data, labels_data,
                                         N, dim, dim, kDimLog2);

  } else {
    ScopedTensorDescriptor desc;
    std::vector<int> tensor_dims = {N, dim, D, 1};
    DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
    miopenTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#else
    hipdnnTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#endif

    auto handle = ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
    auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                 : MIOPEN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::miopenSoftmaxForward_V2(
        handle, platform::CudnnDataType<T>::kOne(), descp, logits_data,
        platform::CudnnDataType<T>::kZero(), descp, softmax_data,
        MIOPEN_SOFTMAX_LOG, mode));
#else
    auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                 : HIPDNN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_LOG, mode, platform::CudnnDataType<T>::kOne(),
        descp, logits_data, platform::CudnnDataType<T>::kZero(), descp,
        softmax_data));
#endif

    const int kDimLog2 = static_cast<int>(Log2Ceil(dim));
    const int kDimCeil = 1 << kDimLog2;
#ifdef __HIPCC__
    int kThreadPerBlock = 256;
#else
    int kThreadPerBlock = 512;
#endif

    int kBatchPerBlock = 1;
    int blocks = (N * D + kBatchPerBlock - 1) / kBatchPerBlock;
    dim3 threads(kThreadPerBlock / kBatchPerBlock, kBatchPerBlock, 1);

    CrossEntropySoftLabel<T, T, true><<<blocks, threads, 0, stream>>>(
        loss_data, softmax_data, NULL, labels_data, N, dim, D, kDimLog2);
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels, const int64_t n,
                                               const int64_t d,
                                               const int64_t remain) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int64_t idx_n = ids / d;
    int64_t idx_remain = ids % remain;
    int64_t idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (logit_grad[ids] - labels[ids]);
  }
}

template <typename T>
__global__ void SoftLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const T* loss_grad,
                                                    const T* labels,
                                                    const int n, const int d,
                                                    const int remain) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int idx_n = ids / d;
    int idx_remain = ids % remain;
    int idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (-labels[ids] / logit_grad[ids]);
  }
}

template <typename T, typename LabelT>
__global__ void HardLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const LabelT* labels,
                                                    const int n, const int d,
                                                    const int remain,
                                                    const int ignore_index) {
  CUDA_KERNEL_LOOP(index, n * remain) {
    int idx_n = index / remain;
    int idx_remain = index % remain;
    int tmp = static_cast<int>(labels[index]);
    int idx = idx_n * d + tmp * remain + idx_remain;
    if (ignore_index != tmp) {
      logit_grad[idx] = -static_cast<T>(1.) / logit_grad[idx];
    }
  }
}

template <typename T, typename LabelT>
__global__ void ScaleCrossEntropyGradient(T* logit_grad, const T* loss_grad,
                                          const int num, const int d,
                                          const int remain,
                                          const LabelT* labels,
                                          const int ignore_index) {
  CUDA_KERNEL_LOOP(index, num) {
    int idx_n = index / d;
    int idx_remain = index % remain;
    int idx_lbl = idx_n * remain + idx_remain;
    int k = (index % d) / remain;
    auto lbl = static_cast<int64_t>(labels[idx_lbl]);
    if (lbl == ignore_index || lbl != k) {
      logit_grad[index] = static_cast<T>(0.);
    } else {
      logit_grad[index] *= loss_grad[idx_lbl];
    }
  }
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    RunSoftmaxWithCrossEntropyFunctor<T>(context, *this);
  }

  template <typename LabelT>
  static void Apply(const framework::ExecutionContext& context,
                    const framework::Tensor& labels, const bool soft_label) {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::Unavailable("softmax_with_cross_entropy operator's "
                                      "CUDA kernel only runs on GPU device."));
    const bool use_softmax = context.Attr<bool>("use_softmax");

    // do not with softmax op, and input is softmax
    if (!use_softmax) {
      const Tensor* softmax = context.Input<Tensor>("Logits");
      Tensor* softmax_out = context.Output<Tensor>("Softmax");
      Tensor* loss = context.Output<Tensor>("Loss");

      const int rank = softmax->dims().size();
      const int axis =
          phi::funcs::CanonicalAxis(context.Attr<int>("axis"), rank);
      const int axis_dim = softmax->dims()[axis];

      const int n = phi::funcs::SizeToAxis(axis, softmax->dims());
      const int d = phi::funcs::SizeFromAxis(axis, softmax->dims());

      auto* softmax_out_data =
          softmax_out->template mutable_data<T>(context.GetPlace());
      auto* loss_data = loss->template mutable_data<T>(context.GetPlace());

      phi::funcs::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      if (axis_dim == 1) {
        set_constant(context.cuda_device_context(), softmax_out,
                     static_cast<T>(1));
        return;
      }

      auto ignore_index = context.Attr<int>("ignore_index");

      Tensor softmax_2d, labels_2d, loss_2d, softmax_out_2d;
      softmax_2d.ShareDataWith(*softmax).Resize({n, d});
      labels_2d.ShareDataWith(labels).Resize({n, labels.numel() / n});
      loss_2d.ShareDataWith(*loss).Resize({n, 1});
      softmax_out_2d.ShareDataWith(*softmax_out).Resize({n, d});

      // math::CrossEntropyFunctor support axis is the last
      if (axis == -1) {
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            soft_label, ignore_index, axis_dim);
        return;
      }

      // if axis is not the last, we need a new impliment
      if (soft_label) {
        auto* logits_data = softmax->template data<T>();
        auto* labels_data = labels.template data<T>();

        const int kDimLog2 = static_cast<int>(Log2Ceil(axis_dim));
        const int kDimCeil = 1 << kDimLog2;
#ifdef __HIPCC__
        int kThreadPerBlock = 256;
#else
        int kThreadPerBlock = 512;
#endif
        int kBatchPerBlock = 1;
        int blocks = (n * d + kBatchPerBlock - 1) / kBatchPerBlock;
        dim3 threads(kThreadPerBlock / kBatchPerBlock, kBatchPerBlock, 1);

        CrossEntropySoftLabel<T, T, false><<<
            blocks, threads, 0, context.cuda_device_context().stream()>>>(
            loss_data, NULL, logits_data, labels_data, n, axis_dim,
            d / axis_dim, kDimLog2);
      } else {  // HardLabel
        auto* logits_data = softmax->template data<T>();
        auto* labels_data = labels.template data<LabelT>();
        int threads = 128;
        int blocks = (n * d / axis_dim + threads - 1) / threads;
        if (ignore_index >= 0 && ignore_index < axis_dim) {
          CrossEntropyHardLabel<T, LabelT, true><<<
              blocks, threads, 0, context.cuda_device_context().stream()>>>(
              loss_data, logits_data, labels_data, n, axis_dim, d / axis_dim,
              ignore_index);
        } else {
          CrossEntropyHardLabel<T, LabelT, false><<<
              blocks, threads, 0, context.cuda_device_context().stream()>>>(
              loss_data, logits_data, labels_data, n, axis_dim, d / axis_dim,
              ignore_index);
        }
      }

      // cause of input is softmax
      // copy to output softmax, directly
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), softmax_out);

      return;
    }

    const Tensor* logits = context.Input<Tensor>("Logits");
    Tensor* softmax = context.Output<Tensor>("Softmax");
    Tensor* loss = context.Output<Tensor>("Loss");

    const int rank = logits->dims().size();
    const int axis = phi::funcs::CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logits->dims()[axis];

    const int64_t n = phi::funcs::SizeToAxis(axis, logits->dims());
    const int64_t d = phi::funcs::SizeFromAxis(axis, logits->dims());

    auto* softmax_data = softmax->template mutable_data<T>(context.GetPlace());
    auto* loss_data = loss->template mutable_data<T>(context.GetPlace());

    if (axis_dim == 1) {
      phi::funcs::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), softmax, static_cast<T>(1));
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      return;
    }

    auto ignore_index = context.Attr<int>("ignore_index");

    if (soft_label) {
      auto* logits_data = logits->template data<T>();
      auto* labels_data = labels.template data<T>();
      SoftmaxWithCrossEntropySoftLabel<T>(
          context.cuda_device_context(), rank, axis, logits_data, labels_data,
          softmax_data, loss_data, n, axis_dim, d / axis_dim);
    } else {
      if (!context.Attr<bool>("numeric_stable_mode")) {
        // CUDNN kernel only suppoer 2-D tensor and perfome softmax on last dim
        Tensor logits_2d, softmax_2d, labels_2d, loss_2d;
        logits_2d.ShareDataWith(*logits).Resize({n, d});
        softmax_2d.ShareDataWith(*softmax).Resize({n, d});
        labels_2d.ShareDataWith(labels).Resize({n, labels.numel() / n});
        loss_2d.ShareDataWith(*loss).Resize({n, 1});
        math::SoftmaxCUDNNFunctor<T>()(context.cuda_device_context(),
                                       &logits_2d, &softmax_2d);
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            false, ignore_index, axis_dim);
      } else {
        auto* logits_data = logits->template data<T>();
        auto* labels_data = labels.template data<LabelT>();
        if (ignore_index >= 0 && ignore_index < axis_dim) {
          SoftmaxWithCrossEntropyHardLabel<T, LabelT, true>(
              context.cuda_device_context(), rank, axis, logits_data,
              labels_data, loss_data, softmax_data, n, axis_dim, d / axis_dim,
              ignore_index);
        } else {
          SoftmaxWithCrossEntropyHardLabel<T, LabelT, false>(
              context.cuda_device_context(), rank, axis, logits_data,
              labels_data, loss_data, softmax_data, n, axis_dim, d / axis_dim,
              ignore_index);
        }
      }
    }
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    RunSoftmaxWithCrossEntropyFunctor<T>(context, *this);
  }

  template <typename LabelT>
  static void Apply(const framework::ExecutionContext& context,
                    const framework::Tensor& labels, const bool soft_label) {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::Unavailable("softmax_with_cross_entropy operator's "
                                      "CUDA kernel only runs on GPU device."));
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))
            ->template data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    const Tensor* softmax = context.Input<Tensor>("Softmax");
    if (logit_grad != softmax) {
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), logit_grad);
    }
    T* logit_grad_data = logit_grad->template data<T>();

    const int rank = logit_grad->dims().size();
    const int axis = phi::funcs::CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logit_grad->dims()[axis];

    const int64_t n = phi::funcs::SizeToAxis(axis, logit_grad->dims());
    const int64_t d = phi::funcs::SizeFromAxis(axis, logit_grad->dims());
    const int64_t remain = d / axis_dim;

#ifdef __HIPCC__
    int block = 256;
#else
    int block = 512;
#endif
    auto stream = context.cuda_device_context().stream();
    auto ignore_index = context.Attr<int>("ignore_index");
    auto use_softmax = context.Attr<bool>("use_softmax");

    // do not with softmax op, and input is softmax
    if (!use_softmax) {
      if (soft_label) {
        int grid = (n * d + block - 1) / block;
        const T* label_data = labels.template data<T>();
        SoftLabelCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
            logit_grad_data, loss_grad_data, label_data, n, d, remain);
      } else {
        Tensor logits_grad_2d;
        logits_grad_2d.ShareDataWith(*logit_grad).Resize({n, d});
        int grid = (n * remain + block - 1) / block;
        const auto* label_data = labels.template data<LabelT>();
        HardLabelCrossEntropyGradientKernel<T,
                                            LabelT><<<grid, block, 0, stream>>>(
            logit_grad_data, label_data, n, d, remain, ignore_index);
        int num = n * d;
        grid = (num + block - 1) / block;
        ScaleCrossEntropyGradient<T, LabelT><<<grid, block, 0, stream>>>(
            logit_grad_data, loss_grad_data, num, d, remain, label_data,
            ignore_index);
      }

      return;
    }

    // with softmax, continue

    if (soft_label) {
      int64_t grid = (n * d + block - 1) / block;
      const T* label_data = labels.template data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d, remain);
    } else {
      const auto* label_data = labels.template data<LabelT>();
      int grid = (n * d + block - 1) / block;
      SoftmaxWithCrossEntropyGradHardLabel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d / remain, remain,
          ignore_index);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>);
#else
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<double>);
#endif
