#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU

#include "paddle/fluid/operators/softmax_with_cross_entropy_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

namespace {
template <typename T>
__global__ void CrossEntropyGrad(T* logit_grad, const int64_t* labels,
                                 const int batch_size, const int class_num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size;
       i += blockDim.x * gridDim.x) {
    int idx = i * class_num + labels[i];
    logit_grad[idx] -= static_cast<T>(1.);
  }
}

template <typename T>
__global__ void Scale(T* logit_grad, const T* loss_grad, const int num,
                      const int class_num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    logit_grad[i] *= loss_grad[i / class_num];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels,
                                               const int batch_size,
                                               const int class_num) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < batch_size * class_num) {
    int row_ids = ids / class_num;
    logit_grad[ids] = loss_grad[row_ids] * (logit_grad[ids] - labels[ids]);
  }
}
}  // namespace

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Label");
    Tensor* softmax = context.Output<Tensor>("Softmax");

    Tensor* loss = context.Output<Tensor>("Loss");
    softmax->mutable_data<T>(context.GetPlace());
    loss->mutable_data<T>(context.GetPlace());

    math::SoftmaxFunctor<platform::CUDADeviceContext, T>()(
        context.cuda_device_context(), logits, softmax);
    math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
        context.cuda_device_context(), loss, softmax, labels,
        context.Attr<bool>("soft_label"));
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    logit_grad->ShareDataWith(*context.Input<Tensor>("Softmax"));
    T* logit_grad_data = logit_grad->data<T>();

    const int batch_size = logit_grad->dims()[0];
    const int class_num = logit_grad->dims()[1];
    int block = 512;
    auto stream = context.cuda_device_context().stream();

    if (context.Attr<bool>("soft_label")) {
      int grid = (batch_size * class_num + block - 1) / block;
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, batch_size, class_num);
    } else {
      int grid = (batch_size + block - 1) / block;
      const int64_t* label_data = labels->data<int64_t>();
      CrossEntropyGrad<T><<<grid, block, 0, stream>>>(
          logit_grad_data, label_data, batch_size, class_num);
      int num = batch_size * class_num;
      grid = (num + block - 1) / block;
      Scale<T><<<grid, block, 0, stream>>>(logit_grad_data, loss_grad_data, num,
                                           class_num);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(softmax_with_cross_entropy,
                        ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
                        ops::SoftmaxWithCrossEntropyCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(softmax_with_cross_entropy_grad,
                        ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
                        ops::SoftmaxWithCrossEntropyGradCUDAKernel<double>);
