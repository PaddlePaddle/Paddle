#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <hipcub/hipcub.hpp>
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/softmax_with_cross_entropy_op.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

namespace {
template <typename T>
__global__ void CrossEntropyGrad(T* logit_grad, const int64_t* labels,
                                 const int n, const int d, const int remain,
                                 const int ignore_index) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n * remain;
       i += blockDim.x * gridDim.x) {
    int idx_n = i / remain;
    int idx_remain = i % remain;
    int idx = idx_n * d + labels[i] * remain + idx_remain;
    logit_grad[idx] -=
        ignore_index == labels[i] ? static_cast<T>(0.) : static_cast<T>(1.);
  }
}

template <typename T>
__global__ void Scale(T* logit_grad, const T* loss_grad, const int num,
                      const int d, const int remain) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    int idx_n = i / d;
    int idx_remain = i % remain;
    logit_grad[i] *= loss_grad[idx_n * remain + idx_remain];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels, const int n,
                                               const int d, const int remain) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int idx_n = ids / d;
    int idx_remain = ids % remain;
    int idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (logit_grad[ids] - labels[ids]);
  }
}

}  // namespace

static __device__ __forceinline__ platform::float16 exp_on_device(
    platform::float16 x) {
  return ::Eigen::numext::exp(x);
}
static __device__ __forceinline__ float exp_on_device(float x) {
  return expf(x);
}
static __device__ __forceinline__ double exp_on_device(double x) {
  return exp(x);
}
static __device__ __forceinline__ platform::float16 log_on_device(
    platform::float16 x) {
  return math::TolerableValue<platform::float16>()(::Eigen::numext::log(x));
}
static __device__ __forceinline__ float log_on_device(float x) {
  return math::TolerableValue<float>()(logf(x));
}
static __device__ __forceinline__ double log_on_device(double x) {
  return math::TolerableValue<double>()(log(x));
}

/** In the following codes, 3 CUDA kernels are implemented to calculate softmax
 * and loss **/
/*
  Supposing the x is `logits` and y is `labels`, the equations are as
followings:
  cross\_entropy_i = \sum_{j}[- y_i_j * log({e^{x_i_j}/\sum_{j}e^{x_i_j}})]
        = \sum_{j}[- y_i_j * log({e^{x_i_j - max_i}/\sum_{j}e^{x_i_j-max_i}})]
        = \sum_{j}[-y_i_j * (x_i_j - max_i - log\sum_{j}e^{x_i_j - max_i})]
        = \sum_{j}[-y_i_j * (x_i_j - max_i - logDiffMaxSum_i)]
        = \sum_{j}(-y_i_j * tmp_i_j)
  softmax_i_j = e^{tmp_i_j}
where:
  max_i = \max_{j}{x_i_j}
  logDiffMaxSum_i = log\sum_{j}e^{x_i_j - max_i}
  tmp_i_j = x_i_j - max_i - logDiffMaxSum_i
Therefore, the calculation can be separated into 3 steps:
Step 1: row-wise operation to calculate max_i
Step 2: row-wise operation to calculate logDiffMaxSum_i
Step 3: caculate tmp_i_j, and finally get softmax_i_j and cross\_entropy_i
To save memory, we can share memory among max_i, logDiffMaxSum_i and
cross\_entropy_i.
In this way, the 3 steps should be changed to:
Step 1 (RowReductionForMax): row-wise operation to calculate max_i
Step 2 (RowReductionForDiffMaxSum): calculate immediate result of softmax'_i_j =
x_i_j - max_i, and row-wise operation to calculate logDiffMaxSum_i
Step 3 (RowReductionForSoftmaxAndCrossEntropy): calculate tmp_i_j = softmax'_i_j
- logDiffMaxSum_i, and finally get softmax_i_j and cross\_entropy_i
*/

// There are 3 kinds of reduce algorithms in cub:
// BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY
// BLOCK_REDUCE_RAKING
// BLOCK_REDUCE_WARP_REDUCTIONS (default)
template <typename T, int BlockDim>
using BlockReduce =
    hipcub::BlockReduce<T, BlockDim /*, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS*/>;

template <typename T, int BlockDim>
using BlockReduceTempStorage = typename BlockReduce<T, BlockDim>::TempStorage;

// Make sure that BlockDim <= axis_dim
// This kernel is used to calculate the max element of each row
template <typename T, int BlockDim>
static __global__ void RowReductionForMax(const T* logits_data, T* max_data,
                                          int d, int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits_data view as [n, axis_dim, remain]
  // max_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int remain = d / axis_dim;
  int idx_n = blockIdx.x / remain;
  int idx_remain = blockIdx.x % remain;
  int beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int end_idx = (idx_n + 1) * d;

  int step = BlockDim * remain;
  T cur_max = logits_data[beg_idx];
  beg_idx += step;
  while (beg_idx < end_idx) {
    if (cur_max < logits_data[beg_idx]) {
      cur_max = logits_data[beg_idx];
    }
    beg_idx += step;
  }

  cur_max = BlockReduce<T, BlockDim>(temp_storage).Reduce(cur_max, hipcub::Max());

  if (threadIdx.x == 0) {
    max_data[blockIdx.x] =
        cur_max < static_cast<T>(-64) ? static_cast<T>(-64) : cur_max;
  }
}

// Make sure that BlockDim <= axis_dim
template <typename T, int BlockDim, bool CalculateLogSoftmax = false>
static __global__ void RowReductionForDiffMaxSum(const T* logits_data,
                                                 T* max_data, T* softmax, int d,
                                                 int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits, softmax data view as [n, axis_dim, remain]
  // max_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int remain = d / axis_dim;
  int idx_n = blockIdx.x / remain;
  int idx_remain = blockIdx.x % remain;
  int beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int end_idx = (idx_n + 1) * d;

  auto block_max = max_data[blockIdx.x];
  int step = BlockDim * remain;

  softmax[beg_idx] = logits_data[beg_idx] - block_max;
  T diff_max_sum = exp_on_device(softmax[beg_idx]);
  auto idx = beg_idx + step;
  while (idx < end_idx) {
    softmax[idx] = logits_data[idx] - block_max;
    diff_max_sum += exp_on_device(softmax[idx]);
    idx += step;
  }

  diff_max_sum =
      BlockReduce<T, BlockDim>(temp_storage).Reduce(diff_max_sum, hipcub::Sum());
  if (threadIdx.x == 0) max_data[blockIdx.x] = log_on_device(diff_max_sum);

  if (!CalculateLogSoftmax) return;
  __syncthreads();
  diff_max_sum = max_data[blockIdx.x];
  softmax[beg_idx] -= diff_max_sum;
  beg_idx += step;
  while (beg_idx < end_idx) {
    softmax[beg_idx] -= diff_max_sum;
    beg_idx += step;
  }
  if (threadIdx.x == 0) max_data[blockIdx.x] = 0;
}

// Make sure that BlockDim <= axis_dim
template <typename T, int BlockDim>
static __global__ void RowReductionForSoftmaxAndCrossEntropy(
    const T* logits_data, const T* labels_data, T* loss_data, T* softmax, int d,
    int axis_dim) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  // logits, softmax, labels data view as [n, axis_dim, remain]
  // loss_data view as [n, 1, remain]
  // blockDim = n * remain, split blockIdx to idx_n and idx_remain
  int remain = d / axis_dim;
  int idx_n = blockIdx.x / remain;
  int idx_remain = blockIdx.x % remain;
  int beg_idx = idx_n * d + threadIdx.x * remain + idx_remain;
  int end_idx = (idx_n + 1) * d;

  // log_diff_max_sum shares memory with loss
  auto block_log_diff_max_sum = loss_data[blockIdx.x];
  auto tmp = softmax[beg_idx] - block_log_diff_max_sum;
  softmax[beg_idx] = exp_on_device(tmp);
  auto loss = -labels_data[beg_idx] * tmp;
  int step = BlockDim * remain;
  beg_idx += step;
  while (beg_idx < end_idx) {
    tmp = softmax[beg_idx] - block_log_diff_max_sum;
    softmax[beg_idx] = exp_on_device(tmp);
    loss -= (labels_data[beg_idx] * tmp);
    beg_idx += step;
  }

  loss = BlockReduce<T, BlockDim>(temp_storage).Reduce(loss, hipcub::Sum());
  if (threadIdx.x == 0) loss_data[blockIdx.x] = loss;
}

template <typename T>
struct HardLabelSoftmaxWithCrossEntropyFunctor {
 public:
  HardLabelSoftmaxWithCrossEntropyFunctor(const int64_t* labels, T* loss,
                                          T* log_softmax, int d, int axis_dim)
      : labels_(labels),
        loss_(loss),
        log_softmax_(log_softmax),
        d_(d),
        axis_dim_(axis_dim) {}

  __device__ void operator()(int idx) const {
    // logits view as [n, axis_dim, remain], where d = axis_dim * remain
    int remain = d_ / axis_dim_;
    int idx_n = idx / d_;
    int idx_axis = (idx % d_) / remain;
    int idx_remain = idx % remain;
    // labels, loss view as [n, remain]
    int idx_lbl = idx_n * remain + idx_remain;
    if (idx_axis != labels_[idx_lbl]) {
      log_softmax_[idx] = exp_on_device(log_softmax_[idx]);
    } else {
      auto softmax = log_softmax_[idx];
      log_softmax_[idx] = exp_on_device(softmax);
      loss_[idx_lbl] = -softmax;
    }
  }

 private:
  const int64_t* labels_;
  T* loss_;
  T* log_softmax_;
  int d_;
  int axis_dim_;
};

template <typename T>
struct HardLabelSoftmaxWithCrossEntropyFunctorWithIgnoreIdx {
 public:
  HardLabelSoftmaxWithCrossEntropyFunctorWithIgnoreIdx(const int64_t* labels,
                                                       T* loss, T* log_softmax,
                                                       int d, int axis_dim,
                                                       int ignore_idx)
      : labels_(labels),
        loss_(loss),
        log_softmax_(log_softmax),
        d_(d),
        axis_dim_(axis_dim),
        ignore_idx_(ignore_idx) {}

  __device__ void operator()(int idx) const {
    // logits view as [n, axis_dim, remain], where d = axis_dim * remain
    int remain = d_ / axis_dim_;
    int idx_n = idx / d_;
    int idx_axis = (idx % d_) / remain;
    int idx_remain = idx % remain;
    // labels, loss view as [n, remain]
    int idx_lbl = idx_n * remain + idx_remain;
    if (idx_axis != labels_[idx_lbl] || idx_axis == ignore_idx_) {
      log_softmax_[idx] = exp_on_device(log_softmax_[idx]);
    } else {
      auto softmax = log_softmax_[idx];
      log_softmax_[idx] = exp_on_device(softmax);
      loss_[idx_lbl] = -softmax;
    }
  }

 private:
  const int64_t* labels_;
  T* loss_;
  T* log_softmax_;
  int d_;
  int axis_dim_;
  int ignore_idx_;
};

template <typename T>
static void HardLabelSoftmaxWithCrossEntropy(
    const platform::CUDADeviceContext& ctx, const T* logits_data,
    const int64_t* labels_data, T* loss_data, T* softmax_data, int n, int d,
    int axis_dim, int ignore_idx) {
  constexpr int kMaxBlockDim = 512;
  int block_dim = axis_dim >= kMaxBlockDim
                      ? kMaxBlockDim
                      : (1 << static_cast<int>(std::log2(axis_dim)));
  int grid_dim = n * d / axis_dim;
  auto stream = ctx.stream();

#define CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)  \
  case BlockDim: {                                                         \
    RowReductionForMax<T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>(    \
        logits_data, loss_data, d, axis_dim);                              \
    RowReductionForDiffMaxSum<T, BlockDim,                                 \
                              true><<<grid_dim, BlockDim, 0, stream>>>(    \
        logits_data, loss_data, softmax_data, d, axis_dim);                \
    platform::ForRange<platform::CUDADeviceContext> for_range(ctx, n* d);  \
    if (ignore_idx >= 0 && ignore_idx < axis_dim) {                        \
      for_range(HardLabelSoftmaxWithCrossEntropyFunctorWithIgnoreIdx<T>(   \
          labels_data, loss_data, softmax_data, d, axis_dim, ignore_idx)); \
    } else {                                                               \
      for_range(HardLabelSoftmaxWithCrossEntropyFunctor<T>(                \
          labels_data, loss_data, softmax_data, d, axis_dim));             \
    }                                                                      \
  } break

  switch (block_dim) {
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(512);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(256);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(128);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(64);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(32);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(16);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(8);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(4);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(2);
    default:
      PADDLE_THROW("BlockDim must be 2^n in softmax_with_cross_entropy_op");
      break;
  }
#undef CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL
}

template <typename T>
static void SoftmaxWithCrossEntropyFusedKernel(const T* logits_data,
                                               const T* labels_data,
                                               T* softmax_data, T* loss_data,
                                               int n, int d, int axis_dim,
                                               hipStream_t stream) {
  constexpr int kMaxBlockDim = 512;
  int block_dim = axis_dim >= kMaxBlockDim
                      ? kMaxBlockDim
                      : (1 << static_cast<int>(std::log2(axis_dim)));
  int grid_dim = n * d / axis_dim;

#define CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)                 \
  case BlockDim:                                                               \
    RowReductionForMax<T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>(        \
        logits_data, loss_data, d, axis_dim);                                  \
    RowReductionForDiffMaxSum<T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>( \
        logits_data, loss_data, softmax_data, d, axis_dim);                    \
    RowReductionForSoftmaxAndCrossEntropy<                                     \
        T, BlockDim><<<grid_dim, BlockDim, 0, stream>>>(                       \
        logits_data, labels_data, loss_data, softmax_data, d, axis_dim);       \
    break

  switch (block_dim) {
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(512);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(256);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(128);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(64);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(32);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(16);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(8);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(4);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(2);
    default:
      PADDLE_THROW("BlockDim must be 2^n in softmax_with_cross_entropy_op");
      break;
  }

#undef CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Label");
    Tensor* softmax = context.Output<Tensor>("Softmax");
    Tensor* loss = context.Output<Tensor>("Loss");

    const int rank = logits->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logits->dims()[axis];

    if (axis_dim == 1) {
      math::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), softmax, static_cast<T>(1));
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      return;
    }

    const int n = SizeToAxis(axis, logits->dims());
    const int d = SizeFromAxis(axis, logits->dims());

    auto* softmax_data = softmax->mutable_data<T>(context.GetPlace());
    auto* loss_data = loss->mutable_data<T>(context.GetPlace());

    auto soft_label = context.Attr<bool>("soft_label");
    auto ignore_index = context.Attr<int>("ignore_index");

    if (soft_label) {
      auto* logits_data = logits->data<T>();
      auto* labels_data = labels->data<T>();
      SoftmaxWithCrossEntropyFusedKernel(
          logits_data, labels_data, softmax_data, loss_data, n, d, axis_dim,
          context.cuda_device_context().stream());
    } else {
      if (!context.Attr<bool>("numeric_stable_mode")) {
        // CUDNN kernel only suppoer 2-D tensor and perfome softmax on last dim
        Tensor logits_2d, softmax_2d, labels_2d, loss_2d;
        logits_2d.ShareDataWith(*logits).Resize({n, d});
        softmax_2d.ShareDataWith(*softmax).Resize({n, d});
        labels_2d.ShareDataWith(*labels).Resize({n, labels->numel() / n});
        loss_2d.ShareDataWith(*loss).Resize({n, 1});
        math::SoftmaxCUDNNFunctor<T>()(context.cuda_device_context(),
                                       &logits_2d, &softmax_2d);
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            false, ignore_index, axis_dim);
      } else {
        auto* logits_data = logits->data<T>();
        auto* labels_data = labels->data<int64_t>();
        HardLabelSoftmaxWithCrossEntropy<T>(
            context.cuda_device_context(), logits_data, labels_data, loss_data,
            softmax_data, n, d, axis_dim, ignore_index);
      }
    }
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    const Tensor* softmax = context.Input<Tensor>("Softmax");
    if (logit_grad != softmax) {
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), logit_grad);
    }
    T* logit_grad_data = logit_grad->data<T>();

    const int rank = logit_grad->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logit_grad->dims()[axis];

    const int n = SizeToAxis(axis, logit_grad->dims());
    const int d = SizeFromAxis(axis, logit_grad->dims());
    const int remain = d / axis_dim;

    int block = 512;
    auto stream = context.cuda_device_context().stream();
    auto ignore_index = context.Attr<int>("ignore_index");
    if (context.Attr<bool>("soft_label")) {
      int grid = (n * d + block - 1) / block;
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d, remain);
    } else {
      int grid = (n * remain + block - 1) / block;
      const int64_t* label_data = labels->data<int64_t>();
      CrossEntropyGrad<T><<<grid, block, 0, stream>>>(
          logit_grad_data, label_data, n, d, remain, ignore_index);
      int num = n * d;
      grid = (num + block - 1) / block;
      Scale<T><<<grid, block, 0, stream>>>(logit_grad_data, loss_grad_data, num,
                                           d, remain);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<double>);
