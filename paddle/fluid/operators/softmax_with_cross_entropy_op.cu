#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/softmax_impl.cuh"
#include "paddle/fluid/operators/softmax_with_cross_entropy_op.h"
#include "paddle/fluid/platform/for_range.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/platform/miopen_helper.h"
#else
#include "paddle/fluid/platform/cudnn_helper.h"
#endif

namespace paddle {
namespace operators {

using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
using Tensor = framework::Tensor;

// Wrapper of log function. Use log(float32) for float16
template <typename T>
static __device__ __forceinline__ T Log(T x) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  AccT logx = std::log(static_cast<AccT>(x));
  return math::TolerableValue<T>()(static_cast<T>(logx));
}

// Wrapper of exp function. Use exp(float32) for float16
template <typename T>
static __device__ __forceinline__ T Exp(T x) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  AccT expx = std::exp(static_cast<AccT>(x));
  return math::TolerableValue<T>()(static_cast<T>(expx));
}

// log2(value)
static inline int Log2Ceil(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) ++log2_value;
  return log2_value;
}

enum class SoftmaxMode { kSoftmax, kLogSoftmax, kCrossEntropy };

/*
  Hard label cross entropy.
*/
template <typename T, bool IgnoreIndex>
__global__ void CrossEntropyHardLabel(T* loss, const T* softmax,
                                      const int64_t* labels, const int n,
                                      const int dim, const int d,
                                      const int ignore_idx) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = ids / d;
  int64_t idx_d = ids % d;

  // thread ids compute loss[ids] using softmax[idx]
  if (ids < n * d) {
    int64_t idx = idx_n * dim * d + labels[ids] * d + idx_d;
    if (IgnoreIndex == true) {
      // IgnoreIndex is true
      if (labels[ids] == ignore_idx) {
        loss[ids] = static_cast<T>(0.0);
      } else {
        loss[ids] = -Log(softmax[idx]);
      }
    } else {
      // IgnoreIndex is false
      loss[ids] = -Log(softmax[idx]);
    }
  }
}

/*
  Hard label cross entropy with exp.
  Input: log softmax
  Output: loss and exp(input)
*/
template <typename T, bool IgnoreIndex>
__global__ void CrossEntropyExpHardLabel(T* loss, T* softmax,
                                         const int64_t* labels, const int n,
                                         const int dim, const int d,
                                         const int ignore_idx) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    if (IgnoreIndex == true) {
      // IgnoreIndex is true
      if (idx_dim == labels[ids]) {
        if (labels[ids] == ignore_idx) {
          loss[ids] = static_cast<T>(0.0);
        } else {
          loss[ids] = -softmax[idx];
        }
      }
    } else {
      // IgnoreIndex is false
      if (labels[ids] >= 0 && labels[ids] < dim) {
        if (labels[ids] == idx_dim) {
          loss[ids] = -softmax[idx];
        }
      } else {
        loss[ids] = static_cast<T>(0.0);
      }
    }
    softmax[idx] = Exp(softmax[idx]);
  }
}

/*
  Core function of softmax with cross entropy forward
    - softmax, SoftmaxMode=kSoftmax
    - log softmax, SoftmaxMode=kLogSoftmax
    - softmax with cross entropy hard label, SoftmaxMode=kCrossEntropy
  The computation includes
    - Compute max value: maxvalue_{i} = max_j src_{i,j}
    - Compute sum of exp: s_{i} = sum_{j}{e^{src_{i,j} - maxvalue_{i}}}
    - Compute: softmax_{i,j} = e^{src_{i,j} - maxvalue_{i}} / s_{i}
    - Compute: logsoftmax_{i,j} = src_{i,j} - maxvalue_{i} - log(s_{i})
    - Compute: loss_{i} = -logsoftmax[i,label[i]] (Hard label)
  This computation results from following formula:
    softmax_{i,j} = e^{src_{i,j}} / sum_{j}{e^{src_{i,j}}}
                  = e^{src_{i,j} - maxvalue_{i}}
                    / sum_{j}{e^{src_{i,j} - maxvalue_{i}}}
                  = e^{src_{i,j} - maxvalue_{i}} / s_{i}
    logsoftmax_{i,j} = log(softmax_{i,j})
                     = src_{i,j} - maxvalue_{i} - log(s_{i})
  One warp (32 threads) is used to compute 1 or 2 batch (kBatchSize).
  For reduction max (sum), firstly compute max (sum) to one warp, then use
  shuffle api to compute max (sum) in one warp.
*/
template <typename T, typename VecT, typename AccT, int Log2Elements,
          SoftmaxMode mode, bool IgnoreIndex>
__global__ void WarpSoftmaxForwardHardLabel(T* loss, T* softmax, const T* src,
                                            const int64_t* label,
                                            const int batch_size,
                                            const int stride,
                                            const int element_count,
                                            const int ignore_index) {
  constexpr int kDimCeil = 1 << Log2Elements;
  constexpr int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  constexpr int kVSize = sizeof(VecT) / sizeof(T);
  constexpr int kIterations = kDimCeil / kWarpSize;
  constexpr int kIterationsV =
      (kIterations >= kVSize) ? (kIterations / kVSize) : 1;
  constexpr int kBatchSize = (kDimCeil <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;

  // max index to read
  int idx_max_v[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; i++) {
    int idx_max = ((i + first_batch) < batch_size) ? element_count : 0;
    idx_max_v[i] = idx_max / kVSize;
  }

  // read data from global memory
  AccT srcdata[kBatchSize][kIterationsV][kVSize];

#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
// read data to srcdata: - KVSize==1, - KVSize>1
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      int src_idx = threadIdx.x + it * kWarpSize;
      if (kVSize == 1) {
        if (src_idx < idx_max_v[i]) {
          srcdata[i][it][0] =
              static_cast<AccT>(src[(first_batch + i) * stride + src_idx]);
        } else {
          srcdata[i][it][0] = -std::numeric_limits<AccT>::infinity();
        }
      } else {
        const VecT* src_v =
            reinterpret_cast<const VecT*>(&src[(first_batch + i) * stride]);
        if (src_idx < idx_max_v[i]) {
          VecT srctmp = src_v[src_idx];
          const T* srcinptr = reinterpret_cast<const T*>(&srctmp);
#pragma unroll
          for (int s = 0; s < kVSize; s++) {
            srcdata[i][it][s] = static_cast<AccT>(srcinptr[s]);
          }
        } else {
#pragma unroll
          for (int s = 0; s < kVSize; s++) {
            srcdata[i][it][s] = -std::numeric_limits<AccT>::infinity();
          }
        }
      }
    }
  }

  // compute max value: maxvalue_{i} = max_j src_{i,j}
  AccT max_value[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    // it = 0
    AccT valmax = srcdata[i][0][0];
#pragma unroll
    for (int s = 1; s < kVSize; ++s) {
      valmax = (valmax > srcdata[i][0][s]) ? valmax : srcdata[i][0][s];
    }
    max_value[i] = valmax;

// it = 1, 2, ...
#pragma unroll
    for (int it = 1; it < kIterationsV; ++it) {
      AccT valmax = srcdata[i][it][0];
#pragma unroll
      for (int s = 1; s < kVSize; ++s) {
        valmax = (valmax > srcdata[i][it][s]) ? valmax : srcdata[i][it][s];
      }
      max_value[i] = (max_value[i] > valmax) ? max_value[i] : valmax;
    }
  }
  WarpReduceMax<AccT, kBatchSize, kWarpSize>(max_value);

  // compute sum: s_{i} = sum_{j}{ exp(src_{i,j} - maxvalue_{i} }
  AccT sum[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    // it = 0
    if (mode == SoftmaxMode::kLogSoftmax ||
        mode == SoftmaxMode::kCrossEntropy) {
      sum[i] = std::exp(srcdata[i][0][0] - max_value[i]);
    } else {
      srcdata[i][0][0] = std::exp(srcdata[i][0][0] - max_value[i]);
      sum[i] = srcdata[i][0][0];
    }
#pragma unroll
    for (int s = 1; s < kVSize; ++s) {
      if (mode == SoftmaxMode::kLogSoftmax ||
          mode == SoftmaxMode::kCrossEntropy) {
        sum[i] += std::exp(srcdata[i][0][s] - max_value[i]);
      } else {
        srcdata[i][0][s] = std::exp(srcdata[i][0][s] - max_value[i]);
        sum[i] += srcdata[i][0][s];
      }
    }

// it = 1, 2, ...
#pragma unroll
    for (int it = 1; it < kIterationsV; ++it) {
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (mode == SoftmaxMode::kLogSoftmax ||
            mode == SoftmaxMode::kCrossEntropy) {
          sum[i] += std::exp(srcdata[i][it][s] - max_value[i]);
        } else {
          srcdata[i][it][s] = std::exp(srcdata[i][it][s] - max_value[i]);
          sum[i] += srcdata[i][it][s];
        }
      }
    }
  }
  WarpReduceSum<AccT, kBatchSize, kWarpSize>(sum);

// write data
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    if (mode == SoftmaxMode::kLogSoftmax ||
        mode == SoftmaxMode::kCrossEntropy) {
      sum[i] = std::log(sum[i]);
    }

#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      int idx = threadIdx.x + it * kWarpSize;
      if (kVSize == 1) {  // kVSize==1
        if (idx < idx_max_v[i]) {
          if (mode == SoftmaxMode::kLogSoftmax) {  // log softmax
            softmax[(first_batch + i) * stride + idx] =
                srcdata[i][it][0] - max_value[i] - sum[i];
            // softmax with cross entropy hard label
          } else if (mode == SoftmaxMode::kCrossEntropy) {
            AccT logsoftmax = srcdata[i][it][0] - max_value[i] - sum[i];
            // softmax
            softmax[(first_batch + i) * stride + idx] = std::exp(logsoftmax);
            // label
            int loss_idx = (threadIdx.x + it * kWarpSize) * kVSize;
            if (IgnoreIndex == true) {
              // IgnoreIndex is true
              if (label[first_batch + i] == loss_idx) {
                if (label[first_batch + i] != ignore_index) {
                  loss[first_batch + i] = -logsoftmax;
                } else {
                  loss[first_batch + i] = static_cast<T>(0.0);
                }
              }
            } else {
              // IgnoreIndex is false
              if (label[first_batch + i] >= 0 &&
                  label[first_batch + i] < element_count) {
                if (label[first_batch + i] == loss_idx) {
                  loss[first_batch + i] = -logsoftmax;
                }
              } else {
                loss[first_batch + i] = static_cast<T>(0.0);
              }
            }
          } else {  // softmax
            softmax[(first_batch + i) * stride + idx] =
                srcdata[i][it][0] / sum[i];
          }
        } else {
          break;
        }
      } else {  // KVSize>1
        VecT* softmax_v =
            reinterpret_cast<VecT*>(&softmax[(first_batch + i) * stride]);
        VecT tmpdata;
        T* tmpptr = reinterpret_cast<T*>(&tmpdata);
#pragma unroll
        for (int s = 0; s < kVSize; ++s) {
          if (mode == SoftmaxMode::kLogSoftmax) {  // log softmax
            tmpptr[s] = srcdata[i][it][s] - max_value[i] - sum[i];
            // softmax with cross entropy hard label
          } else if (mode == SoftmaxMode::kCrossEntropy) {
            AccT logsoftmax = srcdata[i][it][s] - max_value[i] - sum[i];
            // softmax
            tmpptr[s] = std::exp(logsoftmax);
            // label
            int loss_idx = (threadIdx.x + it * kWarpSize) * kVSize + s;
            if (IgnoreIndex == true) {
              // IgnoreIndex is true
              if (label[first_batch + i] == loss_idx &&
                  label[first_batch + i] != ignore_index) {
                loss[first_batch + i] = -logsoftmax;
              }
            } else {
              // IgnoreIndex is false
              if (label[first_batch + i] >= 0 &&
                  label[first_batch + i] < element_count) {
                if (label[first_batch + i] == loss_idx) {
                  loss[first_batch + i] = -logsoftmax;
                }
              } else {
                loss[first_batch + i] = static_cast<T>(0.0);
              }
            }
          } else {  // softmax
            tmpptr[s] = srcdata[i][it][s] / sum[i];
          }
        }
        if (idx < idx_max_v[i]) {
          softmax_v[idx] = tmpdata;
        } else {
          break;
        }
      }
    }
  }
}

#define SOFTMAX_WARP_FORWARD_CASE(Log2Elements, VecT, AccT)                   \
  case Log2Elements:                                                          \
    WarpSoftmaxForwardHardLabel<T, VecT, AccT, Log2Elements, mode,            \
                                IgnoreIndex><<<blocks, threads, 0, stream>>>( \
        loss, softmax, src, label, batch_size, stride, element_count,         \
        ignore_index);                                                        \
    break;

/*
  Wrapper of softmax with cross entropy forward hard label.
*/
template <typename T, SoftmaxMode mode, bool IgnoreIndex>
void SwitchWarpSoftmaxForwardHardLabel(T* loss, T* softmax, const T* src,
                                       const int64_t* label,
                                       const int batch_size, const int stride,
                                       const int element_count,
                                       const int ignore_index,
                                       gpuStream_t stream) {
  using AccT = typename details::MPTypeTrait<T>::Type;

  // use 128 threads per block to maximimize gpu utilization
  const int Log2Elements = static_cast<int>(Log2Ceil(element_count));
  const int kDimCeil = 1 << Log2Elements;
  int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  int batches_per_warp = (kDimCeil <= 128) ? 2 : 1;
  constexpr int threads_per_block = 128;
  int warps_per_block = (threads_per_block / kWarpSize);
  int batches_per_block = warps_per_block * batches_per_warp;
  int blocks = (batch_size + batches_per_block - 1) / batches_per_block;
  dim3 threads(kWarpSize, warps_per_block, 1);

  switch (Log2Elements) {
    SOFTMAX_WARP_FORWARD_CASE(0, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(1, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(2, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(3, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(4, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(5, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(6, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(7, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(8, T, AccT);
    SOFTMAX_WARP_FORWARD_CASE(9, T, AccT);
    default:
      break;
  }
}

/*
  Wrapper of softmax with cross entropy hard label.
  - SwitchWarpSoftmaxForwardHardLabel for small size
  - cudnn function for large size
*/
template <typename T, bool IgnoreIndex>
static void SoftmaxWithCrossEntropyHardLabel(
    const platform::CUDADeviceContext& ctx, int rank, int axis,
    const T* logits_data, const int64_t* labels_data, T* loss_data,
    T* softmax_data, int N, int dim, int D, const int ignore_index) {
  auto stream = ctx.stream();
  constexpr int max_dim = 320;
  if (D == 1 && dim <= max_dim) {  // small size
    const SoftmaxMode mode = SoftmaxMode::kCrossEntropy;
    SwitchWarpSoftmaxForwardHardLabel<T, mode, IgnoreIndex>(
        loss_data, softmax_data, logits_data, labels_data, N, dim, dim,
        ignore_index, stream);
  } else {
    ScopedTensorDescriptor desc;
    std::vector<int> tensor_dims = {N, dim, D, 1};
    DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
    miopenTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#else
    hipdnnTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#endif

    auto handle = ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
    auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                 : MIOPEN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxForward_V2(
        handle, platform::CudnnDataType<T>::kOne(), descp, logits_data,
        platform::CudnnDataType<T>::kZero(), descp, softmax_data,
        MIOPEN_SOFTMAX_LOG, mode));
#else
    auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                 : HIPDNN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_LOG, mode, platform::CudnnDataType<T>::kOne(),
        descp, logits_data, platform::CudnnDataType<T>::kZero(), descp,
        softmax_data));
#endif
    int threads = 128;
    int blocks = (N * dim * D + threads - 1) / threads;
    // compute cross entropy, input is log softmax
    CrossEntropyExpHardLabel<T, IgnoreIndex><<<blocks, threads, 0, stream>>>(
        loss_data, softmax_data, labels_data, N, dim, D, ignore_index);
  }
}

/*
  Wrapper of softmax with cross entropy grad hard label.
*/
template <typename T>
__global__ void SoftmaxWithCrossEntropyGradHardLabel(
    T* logits_grad, const T* loss_grad, const int64_t* labels, const int64_t n,
    const int64_t dim, const int64_t d, const int ignore_index) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    if (labels[ids] == ignore_index) {
      logits_grad[idx] = static_cast<T>(0.0);
    } else if (labels[ids] == idx_dim) {
      logits_grad[idx] =
          (logits_grad[idx] - static_cast<T>(1.0)) * loss_grad[ids];
    } else {
      logits_grad[idx] *= loss_grad[ids];
    }
  }
}

/*
  Cross entropy soft label with dynamic size on axis (Log2Elements is varibale).
*/
template <typename T, typename VecT, bool InLogMode = false>
__global__ void CrossEntropySoftLabel(T* loss, T* softmaxwrt, const T* softmax,
                                      const T* labels, const int n,
                                      const int dim, const int d,
                                      int Log2Elements) {
  const int kDimCeil = 1 << Log2Elements;
  const int kVSize = sizeof(VecT) / sizeof(T);

  const int kThreadPerBlock = 512;
  const int kBatchPerBlock = 1;
  const int kWarpSize = 32;  // (dim < 32) ? dim : 32;
  const int kBatchSize = 1;
  const int kThreadPerBatch = kThreadPerBlock / kBatchPerBlock;
  const int kWarpPerBatch = kThreadPerBatch / kWarpSize;

  const int kIterations = (dim + kThreadPerBatch - 1) / kThreadPerBatch;
  const int kIterationsV = (kIterations >= kVSize) ? (kIterations / kVSize) : 1;

  const int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;

  T sum[kBatchSize]{static_cast<T>(0.0)};
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    int ids = first_batch + i;
    if (ids >= n * d) break;
    int idx_n = ids / d;
    int idx_d = ids % d;
#pragma unroll
    for (int it = 0; it < kIterations; ++it) {
      int idx_dim = it * kThreadPerBatch + threadIdx.x;
      int idx = idx_n * dim * d + idx_dim * d + idx_d;

      if (idx_n < n && idx_dim < dim) {
        VecT softmaxdata;
        if (InLogMode == true) {
          softmaxdata = reinterpret_cast<VecT*>(&softmaxwrt[idx])[0];
        } else {
          softmaxdata = reinterpret_cast<const VecT*>(&softmax[idx])[0];
        }
        VecT labelsdata = reinterpret_cast<const VecT*>(&labels[idx])[0];
        T* softmaxptr = reinterpret_cast<T*>(&softmaxdata);
        T* labelsptr = reinterpret_cast<T*>(&labelsdata);
#pragma unroll
        for (int s = 0; s < kVSize; s++) {
          if (InLogMode == true) {
            sum[i] -= softmaxptr[s] * labelsptr[s];
            softmaxptr[s] = Exp(softmaxptr[s]);
          } else {
            sum[i] -= Log(softmaxptr[s]) * labelsptr[s];
          }
        }
        if (InLogMode == true) {
          reinterpret_cast<VecT*>(&softmaxwrt[idx])[0] = softmaxdata;
        }
      }
    }
  }
  WarpReduceSum<T, kBatchSize, kWarpSize>(sum);
  __syncthreads();

  __shared__ T sumshare[kWarpPerBatch][kBatchPerBlock][kBatchSize];
  if (threadIdx.x % kWarpSize == 0) {
#pragma unroll
    for (int i = 0; i < kBatchSize; i++) {
      sumshare[threadIdx.x / kWarpSize][threadIdx.y][i] = sum[i];
    }
  }
  __syncthreads();

  // write
  if (threadIdx.x == 0) {
    for (int i = 0; i < kBatchSize; i++) {
      int ids = first_batch + i;
      if (ids < n * d) {
        loss[ids] = sumshare[0][threadIdx.y][i];
        for (int s = 1; s < kWarpPerBatch; s++) {
          loss[ids] += sumshare[s][threadIdx.y][i];
        }
      }
    }
  }
}

/*
Core function of softmax with cross entropy forward soft label.
The computation includes
  - Compute maximum of batch: maxvalue_{i} = max_j src_{i,j}
  - Compute sum of exp batch: s_{i} = sum_{j}{ exp(src_{i,j} - maxvalue_{i} }
  - Compute: sum of - sum_{j}{ label_{i,j} * (src_{i,j} - maxvalue_{i} -
log(sum[i]))}
One warp (32 threads) is used to compute 1 or 2 batch (kBatchSize).
For reduction max (sum), firstly compute max (sum) to one warp, then use shuffle
api to compute max (sum) in one warp.
*/
template <typename T, typename VecT, typename AccT, int Log2Elements>
__global__ void WarpSoftmaxForwardSoftLabel(T* loss, T* softmax, const T* src,
                                            const T* label,
                                            const int batch_size,
                                            const int stride,
                                            const int element_count) {
  const bool LogMode = true;

  constexpr int kDimCeil = 1 << Log2Elements;
  constexpr int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
  constexpr int kVSize = sizeof(VecT) / sizeof(T);
  constexpr int kIterations = kDimCeil / kWarpSize;
  constexpr int kIterationsV =
      (kIterations >= kVSize) ? (kIterations / kVSize) : 1;
  constexpr int kBatchSize = (kDimCeil <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * kBatchSize;
  int local_batches = batch_size - first_batch;
  if (local_batches > kBatchSize) {
    local_batches = kBatchSize;
  }

  // read data from global memory
  VecT srcdata[kBatchSize][kIterationsV];
  VecT labeldata[kBatchSize][kIterationsV];

  for (int i = 0; i < kBatchSize; ++i) {
    const VecT* src_v =
        reinterpret_cast<const VecT*>(&src[(first_batch + i) * stride]);
    const VecT* label_v =
        reinterpret_cast<const VecT*>(&label[(first_batch + i) * stride]);

    // max index to read
    int idx_max = (i < local_batches) ? element_count : 0;
    int idx_max_v = idx_max / kVSize;

    // read data
    for (int it = 0; it < kIterationsV; ++it) {
      int src_idx = threadIdx.x + it * kWarpSize;
      if (src_idx < idx_max_v) {
        srcdata[i][it] = src_v[src_idx];
        labeldata[i][it] = label_v[src_idx];
      } else {
#pragma unroll
        for (int s = 0; s < kVSize; s++) {
          reinterpret_cast<T*>(&srcdata[i][it])[s] =
              -std::numeric_limits<AccT>::max();
          reinterpret_cast<T*>(&labeldata[i][it])[s] = 0.0;
        }
      }
    }
  }

  // compute max value
  AccT max_value[kBatchSize];
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    max_value[i] = -std::numeric_limits<AccT>::infinity();
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      T* srcptr_v = reinterpret_cast<T*>(&srcdata[i][it]);
      T valmax = srcptr_v[0];
#pragma unroll
      for (int s = 1; s < kVSize; ++s) {
        valmax = (valmax > srcptr_v[s]) ? valmax : srcptr_v[s];
      }
      max_value[i] = (max_value[i] > static_cast<AccT>(valmax))
                         ? max_value[i]
                         : static_cast<AccT>(valmax);
    }
  }
  WarpReduceMax<AccT, kBatchSize, kWarpSize>(max_value);

  // compute sum
  AccT sum[kBatchSize]{0.0};
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      T* srcptr_v = reinterpret_cast<T*>(&srcdata[i][it]);
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (LogMode) {
          sum[i] += std::exp(static_cast<AccT>(srcptr_v[s]) - max_value[i]);
        } else {
          srcptr_v[s] = std::exp(static_cast<AccT>(srcptr_v[s]) - max_value[i]);
          sum[i] += static_cast<AccT>(srcptr_v[s]);
        }
      }
    }
  }
  WarpReduceSum<AccT, kBatchSize, kWarpSize>(sum);

  // log_softmax and loss
  AccT sumloss[kBatchSize]{0.0};
#pragma unroll
  for (int i = 0; i < kBatchSize; ++i) {
    if (i >= local_batches) break;

    VecT* softmax_v =
        reinterpret_cast<VecT*>(&softmax[(first_batch + i) * stride]);

    // max index to write
    int idx_max = (i < local_batches) ? element_count : 0;
    int idx_max_v = idx_max / kVSize;

    if (LogMode) {
      sum[i] = std::log(sum[i]);
    }
#pragma unroll
    for (int it = 0; it < kIterationsV; ++it) {
      T* srcvp = reinterpret_cast<T*>(&srcdata[i][it]);
      T* labelvp = reinterpret_cast<T*>(&labeldata[i][it]);
      VecT tmpv;
      T* tmpvp = reinterpret_cast<T*>(&tmpv);
#pragma unroll
      for (int s = 0; s < kVSize; ++s) {
        if (LogMode) {
          AccT logsoftmax = static_cast<AccT>(srcvp[s]) - max_value[i] - sum[i];
          sumloss[i] -= logsoftmax * static_cast<AccT>(labelvp[s]);
          tmpvp[s] = std::exp(logsoftmax);
        } else {
          tmpvp[s] = static_cast<AccT>(srcvp[s]) / sum[i];
        }
      }

      int idx = threadIdx.x + it * kWarpSize;
      if (idx < idx_max_v) {
        softmax_v[idx] = tmpv;
      }
    }
  }

  // loss
  WarpReduceSum<AccT, kBatchSize, kWarpSize>(sumloss);

  for (int i = 0; i < kBatchSize; i++) {
    if (i >= local_batches) break;
    loss[first_batch + i] = sumloss[i];
  }
}

#define SOFTMAX_WARP_FORWARD_SOFT_CASE(Log2Elements, VecT, AccT)               \
  case Log2Elements:                                                           \
    WarpSoftmaxForwardSoftLabel<T, VecT, AccT,                                 \
                                Log2Elements><<<blocks, threads, 0, stream>>>( \
        loss, softmax, src, label, batch_size, stride, element_count);         \
    break;

/*
  Wrapper of softmax with cross entropy forward soft label.
*/
template <typename T>
void SwitchWarpSoftmaxForwardSoftLabel(const int blocks, const dim3 threads,
                                       gpuStream_t stream, T* loss, T* softmax,
                                       const T* src, const T* label,
                                       const int batch_size, const int stride,
                                       const int element_count,
                                       const int Log2Elements) {
  using AccT = typename details::MPTypeTrait<T>::Type;
  switch (Log2Elements) {
    SOFTMAX_WARP_FORWARD_SOFT_CASE(0, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(1, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(2, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(3, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(4, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(5, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(6, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(7, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(8, T, AccT);
    SOFTMAX_WARP_FORWARD_SOFT_CASE(9, T, AccT);
    default:
      break;
  }
}

template <typename T>
static void SoftmaxWithCrossEntropySoftLabel(
    const platform::CUDADeviceContext& ctx, const int rank, const int axis,
    const T* logits_data, const T* labels_data, T* softmax_data, T* loss_data,
    int N, int dim, int D) {
#ifdef __HIPCC__
  constexpr int kMaxBlockDim = 256;
#else
  constexpr int kMaxBlockDim = 512;
#endif
  int64_t block_dim = dim >= kMaxBlockDim
                          ? kMaxBlockDim
                          : (1 << static_cast<int>(std::log2(dim)));

  int64_t grid_dim = N * D;
  constexpr int max_dim = 320;

  const int kDimLog2 = static_cast<int>(Log2Ceil(dim));
  const int kDimCeil = 1 << kDimLog2;
  auto stream = ctx.stream();

  if (D == 1 && dim <= max_dim) {
    int kWarpSize = (kDimCeil < 32) ? kDimCeil : 32;
    int batches_per_warp = (kDimCeil <= 128) ? 2 : 1;

    // use 128 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 128;
    int warps_per_block = (threads_per_block / kWarpSize);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = (N + batches_per_block - 1) / batches_per_block;
    dim3 threads(kWarpSize, warps_per_block, 1);

    SwitchWarpSoftmaxForwardSoftLabel<T>(blocks, threads, stream, loss_data,
                                         softmax_data, logits_data, labels_data,
                                         N, dim, dim, kDimLog2);

  } else {
    ScopedTensorDescriptor desc;
    std::vector<int> tensor_dims = {N, dim, D, 1};
    DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
    miopenTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#else
    hipdnnTensorDescriptor_t descp = desc.descriptor<T>(layout, tensor_dims);
#endif

    auto handle = ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
    auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                 : MIOPEN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxForward_V2(
        handle, platform::CudnnDataType<T>::kOne(), descp, logits_data,
        platform::CudnnDataType<T>::kZero(), descp, softmax_data,
        MIOPEN_SOFTMAX_LOG, mode));
#else
    auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                 : HIPDNN_SOFTMAX_MODE_CHANNEL;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
        handle, HIPDNN_SOFTMAX_LOG, mode, platform::CudnnDataType<T>::kOne(),
        descp, logits_data, platform::CudnnDataType<T>::kZero(), descp,
        softmax_data));
#endif

    const int kDimLog2 = static_cast<int>(Log2Ceil(dim));
    const int kDimCeil = 1 << kDimLog2;

    int kThreadPerBlock = 512;
    int kBatchPerBlock = 1;
    int blocks = (N * D + kBatchPerBlock - 1) / kBatchPerBlock;
    dim3 threads(kThreadPerBlock / kBatchPerBlock, kBatchPerBlock, 1);

    CrossEntropySoftLabel<T, T, true><<<blocks, threads, 0, stream>>>(
        loss_data, softmax_data, NULL, labels_data, N, dim, D, kDimLog2);
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels, const int64_t n,
                                               const int64_t d,
                                               const int64_t remain) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int64_t idx_n = ids / d;
    int64_t idx_remain = ids % remain;
    int64_t idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (logit_grad[ids] - labels[ids]);
  }
}

template <typename T>
__global__ void SoftLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const T* loss_grad,
                                                    const T* labels,
                                                    const int n, const int d,
                                                    const int remain) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int idx_n = ids / d;
    int idx_remain = ids % remain;
    int idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (-labels[ids] / logit_grad[ids]);
  }
}

template <typename T>
__global__ void HardLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const int64_t* labels,
                                                    const int n, const int d,
                                                    const int remain,
                                                    const int ignore_index) {
  CUDA_KERNEL_LOOP(index, n * remain) {
    int idx_n = index / remain;
    int idx_remain = index % remain;
    int tmp = labels[index];
    int idx = idx_n * d + tmp * remain + idx_remain;
    if (ignore_index != tmp) {
      logit_grad[idx] = -static_cast<T>(1.) / logit_grad[idx];
    }
  }
}

template <typename T>
__global__ void ScaleCrossEntropyGradient(T* logit_grad, const T* loss_grad,
                                          const int num, const int d,
                                          const int remain,
                                          const int64_t* labels,
                                          const int ignore_index) {
  CUDA_KERNEL_LOOP(index, num) {
    int idx_n = index / d;
    int idx_remain = index % remain;
    int idx_lbl = idx_n * remain + idx_remain;
    int k = (index % d) / remain;
    if (labels[idx_lbl] == ignore_index || labels[idx_lbl] != k) {
      logit_grad[index] = static_cast<T>(0.);
    } else {
      logit_grad[index] *= loss_grad[idx_lbl];
    }
  }
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::Unavailable("softmax_with_cross_entropy operator's "
                                      "CUDA kernel only runs on GPU device."));
    const bool use_softmax = context.Attr<bool>("use_softmax");

    // do not with softmax op, and input is softmax
    if (!use_softmax) {
      const Tensor* softmax = context.Input<Tensor>("Logits");
      const Tensor* labels = context.Input<Tensor>("Label");
      Tensor* softmax_out = context.Output<Tensor>("Softmax");
      Tensor* loss = context.Output<Tensor>("Loss");

      const int rank = softmax->dims().size();
      const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
      const int axis_dim = softmax->dims()[axis];

      const int n = SizeToAxis(axis, softmax->dims());
      const int d = SizeFromAxis(axis, softmax->dims());

      auto* softmax_out_data = softmax_out->mutable_data<T>(context.GetPlace());
      auto* loss_data = loss->mutable_data<T>(context.GetPlace());

      math::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      if (axis_dim == 1) {
        set_constant(context.cuda_device_context(), softmax_out,
                     static_cast<T>(1));
        return;
      }

      auto soft_label = context.Attr<bool>("soft_label");
      auto ignore_index = context.Attr<int>("ignore_index");

      Tensor softmax_2d, labels_2d, loss_2d, softmax_out_2d;
      softmax_2d.ShareDataWith(*softmax).Resize({n, d});
      labels_2d.ShareDataWith(*labels).Resize({n, labels->numel() / n});
      loss_2d.ShareDataWith(*loss).Resize({n, 1});
      softmax_out_2d.ShareDataWith(*softmax_out).Resize({n, d});

      // math::CrossEntropyFunctor support axis is the last
      if (axis == -1) {
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            soft_label, ignore_index, axis_dim);
        return;
      }

      // if axis is not the last, we need a new impliment
      if (soft_label) {
        auto* logits_data = softmax->data<T>();
        auto* labels_data = labels->data<T>();

        const int kDimLog2 = static_cast<int>(Log2Ceil(axis_dim));
        const int kDimCeil = 1 << kDimLog2;
        int kThreadPerBlock = 512;
        int kBatchPerBlock = 1;
        int blocks = (n * d + kBatchPerBlock - 1) / kBatchPerBlock;
        dim3 threads(kThreadPerBlock / kBatchPerBlock, kBatchPerBlock, 1);

        CrossEntropySoftLabel<T, T, false><<<
            blocks, threads, 0, context.cuda_device_context().stream()>>>(
            loss_data, NULL, logits_data, labels_data, n, axis_dim,
            d / axis_dim, kDimLog2);
      } else {  // HardLabel
        auto* logits_data = softmax->data<T>();
        auto* labels_data = labels->data<int64_t>();
        int threads = 128;
        int blocks = (n * d / axis_dim + threads - 1) / threads;
        if (ignore_index >= 0 && ignore_index < axis_dim) {
          CrossEntropyHardLabel<T, true><<<
              blocks, threads, 0, context.cuda_device_context().stream()>>>(
              loss_data, logits_data, labels_data, n, axis_dim, d / axis_dim,
              ignore_index);
        } else {
          CrossEntropyHardLabel<T, false><<<
              blocks, threads, 0, context.cuda_device_context().stream()>>>(
              loss_data, logits_data, labels_data, n, axis_dim, d / axis_dim,
              ignore_index);
        }
      }

      // cause of input is softmax
      // copy to output softmax, directly
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), softmax_out);

      return;
    }

    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Label");
    Tensor* softmax = context.Output<Tensor>("Softmax");
    Tensor* loss = context.Output<Tensor>("Loss");

    const int rank = logits->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logits->dims()[axis];

    const int64_t n = SizeToAxis(axis, logits->dims());
    const int64_t d = SizeFromAxis(axis, logits->dims());

    auto* softmax_data = softmax->mutable_data<T>(context.GetPlace());
    auto* loss_data = loss->mutable_data<T>(context.GetPlace());

    if (axis_dim == 1) {
      math::SetConstant<platform::CUDADeviceContext, T> set_constant;
      set_constant(context.cuda_device_context(), softmax, static_cast<T>(1));
      set_constant(context.cuda_device_context(), loss, static_cast<T>(0));
      return;
    }

    auto soft_label = context.Attr<bool>("soft_label");
    auto ignore_index = context.Attr<int>("ignore_index");

    if (soft_label) {
      auto* logits_data = logits->data<T>();
      auto* labels_data = labels->data<T>();
      SoftmaxWithCrossEntropySoftLabel<T>(
          context.cuda_device_context(), rank, axis, logits_data, labels_data,
          softmax_data, loss_data, n, axis_dim, d / axis_dim);
    } else {
      if (!context.Attr<bool>("numeric_stable_mode")) {
        // CUDNN kernel only suppoer 2-D tensor and perfome softmax on last dim
        Tensor logits_2d, softmax_2d, labels_2d, loss_2d;
        logits_2d.ShareDataWith(*logits).Resize({n, d});
        softmax_2d.ShareDataWith(*softmax).Resize({n, d});
        labels_2d.ShareDataWith(*labels).Resize({n, labels->numel() / n});
        loss_2d.ShareDataWith(*loss).Resize({n, 1});
        math::SoftmaxCUDNNFunctor<T>()(context.cuda_device_context(),
                                       &logits_2d, &softmax_2d);
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), &loss_2d, &softmax_2d, &labels_2d,
            false, ignore_index, axis_dim);
      } else {
        auto* logits_data = logits->data<T>();
        auto* labels_data = labels->data<int64_t>();
        if (ignore_index >= 0 && ignore_index < axis_dim) {
          SoftmaxWithCrossEntropyHardLabel<T, true>(
              context.cuda_device_context(), rank, axis, logits_data,
              labels_data, loss_data, softmax_data, n, axis_dim, d / axis_dim,
              ignore_index);
        } else {
          SoftmaxWithCrossEntropyHardLabel<T, false>(
              context.cuda_device_context(), rank, axis, logits_data,
              labels_data, loss_data, softmax_data, n, axis_dim, d / axis_dim,
              ignore_index);
        }
      }
    }
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::Unavailable("softmax_with_cross_entropy operator's "
                                      "CUDA kernel only runs on GPU device."));
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    const Tensor* softmax = context.Input<Tensor>("Softmax");
    if (logit_grad != softmax) {
      framework::TensorCopy(*softmax, context.GetPlace(),
                            context.device_context(), logit_grad);
    }
    T* logit_grad_data = logit_grad->data<T>();

    const int rank = logit_grad->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);
    int axis_dim = logit_grad->dims()[axis];

    const int64_t n = SizeToAxis(axis, logit_grad->dims());
    const int64_t d = SizeFromAxis(axis, logit_grad->dims());
    const int64_t remain = d / axis_dim;

    int block = 512;
    auto stream = context.cuda_device_context().stream();
    auto ignore_index = context.Attr<int>("ignore_index");
    auto use_softmax = context.Attr<bool>("use_softmax");

    // do not with softmax op, and input is softmax
    if (!use_softmax) {
      if (context.Attr<bool>("soft_label")) {
        int grid = (n * d + block - 1) / block;
        const T* label_data = labels->data<T>();
        SoftLabelCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
            logit_grad_data, loss_grad_data, label_data, n, d, remain);
      } else {
        Tensor logits_grad_2d;
        logits_grad_2d.ShareDataWith(*logit_grad).Resize({n, d});
        int grid = (n * remain + block - 1) / block;
        const int64_t* label_data = labels->data<int64_t>();
        HardLabelCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
            logit_grad_data, label_data, n, d, remain, ignore_index);
        int num = n * d;
        grid = (num + block - 1) / block;
        ScaleCrossEntropyGradient<T><<<grid, block, 0, stream>>>(
            logit_grad_data, loss_grad_data, num, d, remain, label_data,
            ignore_index);
      }

      return;
    }

    // with softmax, continue

    if (context.Attr<bool>("soft_label")) {
      int64_t grid = (n * d + block - 1) / block;
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d, remain);
    } else {
      const int64_t* label_data = labels->data<int64_t>();
      int grid = (n * d + block - 1) / block;
      SoftmaxWithCrossEntropyGradHardLabel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d / remain, remain,
          ignore_index);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>);
#else
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy, ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(
    softmax_with_cross_entropy_grad,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<paddle::platform::float16>,
    ops::SoftmaxWithCrossEntropyGradCUDAKernel<double>);
#endif