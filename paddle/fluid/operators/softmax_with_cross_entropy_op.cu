#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU

#include <hipcub/hipcub.hpp>
#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/operators/softmax_with_cross_entropy_op.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

namespace {
template <typename T>
__global__ void CrossEntropyGrad(T* logit_grad, const int64_t* labels,
                                 const int batch_size, const int class_num,
                                 const int ignore_index) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size;
       i += blockDim.x * gridDim.x) {
    int idx = i * class_num + labels[i];
    logit_grad[idx] -=
        ignore_index == labels[i] ? static_cast<T>(0.) : static_cast<T>(1.);
  }
}

template <typename T>
__global__ void Scale(T* logit_grad, const T* loss_grad, const int num,
                      const int class_num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    logit_grad[i] *= loss_grad[i / class_num];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels,
                                               const int batch_size,
                                               const int class_num) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < batch_size * class_num) {
    int row_ids = ids / class_num;
    logit_grad[ids] = loss_grad[row_ids] * (logit_grad[ids] - labels[ids]);
  }
}

}  // namespace

static __device__ __forceinline__ float real_exp(float x) { return expf(x); }
static __device__ __forceinline__ double real_exp(double x) { return exp(x); }
static __device__ __forceinline__ float real_log(float x) {
  return math::TolerableValue<float>()(logf(x));
}
static __device__ __forceinline__ double real_log(double x) {
  return math::TolerableValue<double>()(log(x));
}

/** In the following codes, 3 CUDA kernels are implemented to calculate softmax
 * and loss **/
/*
  Supposing the x is `logits` and y is `labels`, the equations are as
followings:

  cross\_entropy_i = \sum_{j}[- y_i_j * log({e^{x_i_j}/\sum_{j}e^{x_i_j}})]
        = \sum_{j}[- y_i_j * log({e^{x_i_j - max_i}/\sum_{j}e^{x_i_j-max_i}})]
        = \sum_{j}[-y_i_j * (x_i_j - max_i - log\sum_{j}e^{x_i_j - max_i})]
        = \sum_{j}[-y_i_j * (x_i_j - max_i - logDiffMaxSum_i)]
        = \sum_{j}(-y_i_j * tmp_i_j)

  softmax_i_j = e^{tmp_i_j}

where:
  max_i = \max_{j}{x_i_j}
  logDiffMaxSum_i = log\sum_{j}e^{x_i_j - max_i}
  tmp_i_j = x_i_j - max_i - logDiffMaxSum_i

Therefore, the calculation can be separated into 3 steps:
Step 1: row-wise operation to calculate max_i
Step 2: row-wise operation to calculate logDiffMaxSum_i
Step 3: caculate tmp_i_j, and finally get softmax_i_j and cross\_entropy_i

To save memory, we can share memory among max_i, logDiffMaxSum_i and
cross\_entropy_i.
In this way, the 3 steps should be changed to:
Step 1 (RowReductionForMax): row-wise operation to calculate max_i
Step 2 (RowReductionForDiffMaxSum): calculate immediate result of softmax'_i_j =
x_i_j - max_i, and row-wise operation to calculate logDiffMaxSum_i
Step 3 (RowReductionForSoftmaxAndCrossEntropy): calculate tmp_i_j = softmax'_i_j
- logDiffMaxSum_i, and finally get softmax_i_j and cross\_entropy_i
*/

// There are 3 kinds of reduce algorithms in cub:
// BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY
// BLOCK_REDUCE_RAKING
// BLOCK_REDUCE_WARP_REDUCTIONS (default)
template <typename T, int BlockDim>
using BlockReduce =
    hipcub::BlockReduce<T, BlockDim /*, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS*/>;

template <typename T, int BlockDim>
using BlockReduceTempStorage = typename BlockReduce<T, BlockDim>::TempStorage;

// Make sure that BlockDim <= feature_size
// This kernel is used to calculate the max element of each row
template <typename T, int BlockDim>
static __global__ void RowReductionForMax(const T* logits_data, T* max_data,
                                          int feature_size) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  auto beg_idx = feature_size * blockIdx.x + threadIdx.x;
  auto end_idx = feature_size * (blockIdx.x + 1);

  T cur_max = logits_data[beg_idx];
  beg_idx += BlockDim;
  while (beg_idx < end_idx) {
    if (cur_max < logits_data[beg_idx]) {
      cur_max = logits_data[beg_idx];
    }
    beg_idx += BlockDim;
  }

  cur_max = BlockReduce<T, BlockDim>(temp_storage).Reduce(cur_max, hipcub::Max());

  if (threadIdx.x == 0) {
    max_data[blockIdx.x] = cur_max < -64 ? -64 : cur_max;
  }
}

// Make sure that BlockDim <= feature_size
template <typename T, int BlockDim, bool CalculateLogSoftmax = false>
static __global__ void RowReductionForDiffMaxSum(const T* logits_data,
                                                 T* max_data, T* softmax,
                                                 int feature_size) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  auto beg_idx = feature_size * blockIdx.x + threadIdx.x;
  auto end_idx = feature_size * (blockIdx.x + 1);

  auto block_max = max_data[blockIdx.x];

  softmax[beg_idx] = logits_data[beg_idx] - block_max;
  T diff_max_sum = real_exp(softmax[beg_idx]);
  auto idx = beg_idx + BlockDim;
  while (idx < end_idx) {
    softmax[idx] = logits_data[idx] - block_max;
    diff_max_sum += real_exp(softmax[idx]);
    idx += BlockDim;
  }

  diff_max_sum =
      BlockReduce<T, BlockDim>(temp_storage).Reduce(diff_max_sum, hipcub::Sum());
  if (threadIdx.x == 0) max_data[blockIdx.x] = real_log(diff_max_sum);

  if (!CalculateLogSoftmax) return;
  __syncthreads();
  diff_max_sum = max_data[blockIdx.x];
  softmax[beg_idx] -= diff_max_sum;
  beg_idx += BlockDim;
  while (beg_idx < end_idx) {
    softmax[beg_idx] -= diff_max_sum;
    beg_idx += BlockDim;
  }
  if (threadIdx.x == 0) max_data[blockIdx.x] = 0;
}

// Make sure that BlockDim <= feature_size
template <typename T, int BlockDim>
static __global__ void RowReductionForSoftmaxAndCrossEntropy(
    const T* logits_data, const T* labels_data, T* loss_data, T* softmax,
    int feature_size) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;

  auto beg_idx = feature_size * blockIdx.x + threadIdx.x;
  auto end_idx = feature_size * (blockIdx.x + 1);

  // log_diff_max_sum shares memory with loss
  auto block_log_diff_max_sum = loss_data[blockIdx.x];
  auto tmp = softmax[beg_idx] - block_log_diff_max_sum;
  softmax[beg_idx] = real_exp(tmp);
  auto loss = -labels_data[beg_idx] * tmp;
  beg_idx += BlockDim;
  while (beg_idx < end_idx) {
    tmp = softmax[beg_idx] - block_log_diff_max_sum;
    softmax[beg_idx] = real_exp(tmp);
    loss -= (labels_data[beg_idx] * tmp);
    beg_idx += BlockDim;
  }

  loss = BlockReduce<T, BlockDim>(temp_storage).Reduce(loss, hipcub::Sum());
  if (threadIdx.x == 0) loss_data[blockIdx.x] = loss;
}

template <typename T>
struct HardLabelSoftmaxWithCrossEntropyFunctor {
 public:
  HardLabelSoftmaxWithCrossEntropyFunctor(const T* logits,
                                          const int64_t* labels, T* loss,
                                          T* log_softmax, int feature_size)
      : logits_(logits),
        labels_(labels),
        loss_(loss),
        log_softmax_(log_softmax),
        feature_size_(feature_size) {}

  __device__ void operator()(int idx) const {
    auto row_idx = idx / feature_size_;
    auto col_idx = idx % feature_size_;
    if (col_idx != labels_[row_idx]) {
      log_softmax_[idx] = real_exp(log_softmax_[idx]);
    } else {
      auto softmax = log_softmax_[idx];
      log_softmax_[idx] = real_exp(softmax);
      loss_[row_idx] = -softmax;
    }
  }

 private:
  const T* logits_;
  const int64_t* labels_;
  T* loss_;
  T* log_softmax_;
  int feature_size_;
};

template <typename T>
struct HardLabelSoftmaxWithCrossEntropyFunctorWithIgnoreIdx {
 public:
  HardLabelSoftmaxWithCrossEntropyFunctorWithIgnoreIdx(const T* logits,
                                                       const int64_t* labels,
                                                       T* loss, T* log_softmax,
                                                       int feature_size,
                                                       int ignore_idx)
      : logits_(logits),
        labels_(labels),
        loss_(loss),
        log_softmax_(log_softmax),
        feature_size_(feature_size),
        ignore_idx_(ignore_idx) {}

  __device__ void operator()(int idx) const {
    auto row_idx = idx / feature_size_;
    auto col_idx = idx % feature_size_;
    if (col_idx != labels_[row_idx] || col_idx == ignore_idx_) {
      log_softmax_[idx] = real_exp(log_softmax_[idx]);
    } else {
      auto softmax = log_softmax_[idx];
      log_softmax_[idx] = real_exp(softmax);
      loss_[row_idx] = -softmax;
    }
  }

 private:
  const T* logits_;
  const int64_t* labels_;
  T* loss_;
  T* log_softmax_;
  int feature_size_;
  int ignore_idx_;
};

template <typename T>
static __global__ void SetSoftmaxToOneWhenFeatureSizeIsOne(T* out,
                                                           int batch_size) {
  auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < batch_size) out[idx] = static_cast<T>(1);
}

template <typename T>
static void HardLabelSoftmaxWithCrossEntropy(
    const platform::CUDADeviceContext& ctx, const T* logits_data,
    const int64_t* labels_data, T* loss_data, T* softmax_data, int batch_size,
    int feature_size, int ignore_idx) {
  constexpr int kMaxBlockDim = 512;
  int block_dim = feature_size >= kMaxBlockDim
                      ? kMaxBlockDim
                      : (1 << static_cast<int>(std::log2(feature_size)));
  auto stream = ctx.stream();

#define CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)    \
  case BlockDim: {                                                           \
    RowReductionForMax<T, BlockDim><<<batch_size, BlockDim, 0, stream>>>(    \
        logits_data, loss_data, feature_size);                               \
    RowReductionForDiffMaxSum<T, BlockDim,                                   \
                              true><<<batch_size, BlockDim, 0, stream>>>(    \
        logits_data, loss_data, softmax_data, feature_size);                 \
    platform::ForRange<platform::CUDADeviceContext> for_range(               \
        ctx, batch_size* feature_size);                                      \
    if (ignore_idx >= 0 && ignore_idx < feature_size) {                      \
      for_range(HardLabelSoftmaxWithCrossEntropyFunctorWithIgnoreIdx<T>(     \
          logits_data, labels_data, loss_data, softmax_data, feature_size,   \
          ignore_idx));                                                      \
    } else {                                                                 \
      for_range(HardLabelSoftmaxWithCrossEntropyFunctor<T>(                  \
          logits_data, labels_data, loss_data, softmax_data, feature_size)); \
    }                                                                        \
  } break

  switch (block_dim) {
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(512);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(256);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(128);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(64);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(32);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(16);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(8);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(4);
    CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(2);
    case 1:
      SetSoftmaxToOneWhenFeatureSizeIsOne<<<(batch_size + kMaxBlockDim - 1) /
                                                kMaxBlockDim,
                                            kMaxBlockDim, 0, stream>>>(
          softmax_data, batch_size);
      hipMemsetAsync(loss_data, 0, batch_size * sizeof(T), stream);
      break;
    default:
      PADDLE_THROW("BlockDim must be 2^n in softmax_with_cross_entropy_op");
      break;
  }
#undef CALL_HARD_LABEL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL
}

template <typename T>
static void SoftmaxWithCrossEntropyFusedKernel(const T* logits_data,
                                               const T* labels_data,
                                               T* softmax_data, T* loss_data,
                                               int batch_size, int feature_size,
                                               hipStream_t stream) {
  constexpr int kMaxBlockDim = 512;
  int block_dim = feature_size >= kMaxBlockDim
                      ? kMaxBlockDim
                      : (1 << static_cast<int>(std::log2(feature_size)));

#define CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(BlockDim)                \
  case BlockDim:                                                              \
    RowReductionForMax<T, BlockDim><<<batch_size, BlockDim, 0, stream>>>(     \
        logits_data, loss_data, feature_size);                                \
    RowReductionForDiffMaxSum<T,                                              \
                              BlockDim><<<batch_size, BlockDim, 0, stream>>>( \
        logits_data, loss_data, softmax_data, feature_size);                  \
    RowReductionForSoftmaxAndCrossEntropy<                                    \
        T, BlockDim><<<batch_size, BlockDim, 0, stream>>>(                    \
        logits_data, labels_data, loss_data, softmax_data, feature_size);     \
    break

  switch (block_dim) {
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(512);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(256);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(128);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(64);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(32);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(16);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(8);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(4);
    CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL(2);
    case 1:
      SetSoftmaxToOneWhenFeatureSizeIsOne<<<(batch_size + kMaxBlockDim - 1) /
                                                kMaxBlockDim,
                                            kMaxBlockDim, 0, stream>>>(
          softmax_data, batch_size);
      hipMemsetAsync(loss_data, 0, batch_size * sizeof(T), stream);
      break;
    default:
      PADDLE_THROW("BlockDim must be 2^n in softmax_with_cross_entropy_op");
      break;
  }

#undef CALL_SOFTMAX_WITH_CROSS_ENTROPY_FUSED_KERNEL
}

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Label");
    Tensor* softmax = context.Output<Tensor>("Softmax");

    Tensor* loss = context.Output<Tensor>("Loss");
    auto* softmax_data = softmax->mutable_data<T>(context.GetPlace());
    auto* loss_data = loss->mutable_data<T>(context.GetPlace());

    auto soft_label = context.Attr<bool>("soft_label");
    auto ignore_index = context.Attr<int>("ignore_index");
    if (soft_label) {
      int batch_size = logits->dims()[0];
      int feature_size = logits->dims()[1];
      auto* logits_data = logits->data<T>();
      auto* labels_data = labels->data<T>();
      SoftmaxWithCrossEntropyFusedKernel(
          logits_data, labels_data, softmax_data, loss_data, batch_size,
          feature_size, context.cuda_device_context().stream());
    } else {
      if (!context.Attr<bool>("numeric_stable_mode")) {
        math::SoftmaxCUDNNFunctor<T>()(context.cuda_device_context(), logits,
                                       softmax);
        math::CrossEntropyFunctor<platform::CUDADeviceContext, T>()(
            context.cuda_device_context(), loss, softmax, labels, false,
            ignore_index);
      } else {
        int batch_size = logits->dims()[0];
        int feature_size = logits->dims()[1];
        auto* logits_data = logits->data<T>();
        auto* labels_data = labels->data<int64_t>();
        HardLabelSoftmaxWithCrossEntropy<T>(
            context.cuda_device_context(), logits_data, labels_data, loss_data,
            softmax_data, batch_size, feature_size, ignore_index);
      }
    }
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    logit_grad->ShareDataWith(*context.Input<Tensor>("Softmax"));
    T* logit_grad_data = logit_grad->data<T>();

    const int batch_size = logit_grad->dims()[0];
    const int class_num = logit_grad->dims()[1];
    int block = 512;
    auto stream = context.cuda_device_context().stream();
    auto ignore_index = context.Attr<int>("ignore_index");
    if (context.Attr<bool>("soft_label")) {
      int grid = (batch_size * class_num + block - 1) / block;
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, batch_size, class_num);
    } else {
      int grid = (batch_size + block - 1) / block;
      const int64_t* label_data = labels->data<int64_t>();
      CrossEntropyGrad<T><<<grid, block, 0, stream>>>(
          logit_grad_data, label_data, batch_size, class_num, ignore_index);
      int num = batch_size * class_num;
      grid = (num + block - 1) / block;
      Scale<T><<<grid, block, 0, stream>>>(logit_grad_data, loss_grad_data, num,
                                           class_num);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(softmax_with_cross_entropy,
                        ops::SoftmaxWithCrossEntropyCUDAKernel<float>,
                        ops::SoftmaxWithCrossEntropyCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(softmax_with_cross_entropy_grad,
                        ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>,
                        ops::SoftmaxWithCrossEntropyGradCUDAKernel<double>);
