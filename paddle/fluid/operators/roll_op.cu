#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/roll_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T>
__global__ void roll_cuda_kernel(const T* input, T* output, int64_t N,
                                 int64_t* shifts, int64_t* strides,
                                 int64_t* sizes, int64_t nums) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }
  int64_t output_idx = idx;
  int64_t dim_idx, dim_idx_shift;
  for (int64_t i = 0; i < nums; i++) {
    dim_idx = idx % (strides[i] * sizes[i]) / strides[i];
    dim_idx_shift = (dim_idx + shifts[i]) % sizes[i];
    output_idx = output_idx + (dim_idx_shift - dim_idx) * strides[i];
  }
  output[output_idx] = input[idx];
}

template <typename DeviceContext, typename T>
class RollCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = framework::stride(input_dim);

    int64_t dim, size;
    size_t gpu_memory_size_ = sizeof(int64_t) * nums;
    std::vector<int64_t> strides, sizes;
    strides.resize(nums);
    sizes.resize(nums);
    paddle::memory::AllocationPtr shifts_gpu =
        memory::Alloc(context.GetPlace(), gpu_memory_size_);
    paddle::memory::AllocationPtr strides_gpu =
        memory::Alloc(context.GetPlace(), gpu_memory_size_);
    paddle::memory::AllocationPtr sizes_gpu =
        memory::Alloc(context.GetPlace(), gpu_memory_size_);

    for (size_t i = 0; i < nums; i++) {
      dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
      size = input_dim[dim];
      shifts[i] = (shifts[i] % size + size) % size;
      strides[i] = stride_dim[dim];
      sizes[i] = size;
    }
    paddle::memory::Copy(
        BOOST_GET_CONST(platform::CUDAPlace, shifts_gpu->place()),
        shifts_gpu->ptr(), platform::CPUPlace(), shifts.data(),
        gpu_memory_size_, stream);
    paddle::memory::Copy(
        BOOST_GET_CONST(platform::CUDAPlace, strides_gpu->place()),
        strides_gpu->ptr(), platform::CPUPlace(), strides.data(),
        gpu_memory_size_, stream);
    paddle::memory::Copy(
        BOOST_GET_CONST(platform::CUDAPlace, sizes_gpu->place()),
        sizes_gpu->ptr(), platform::CPUPlace(), sizes.data(), gpu_memory_size_,
        stream);
    int64_t* shifts_ptr = reinterpret_cast<int64_t*>(shifts_gpu->ptr());
    int64_t* strides_ptr = reinterpret_cast<int64_t*>(strides_gpu->ptr());
    int64_t* sizes_ptr = reinterpret_cast<int64_t*>(sizes_gpu->ptr());

    roll_cuda_kernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                           PADDLE_CUDA_NUM_THREADS,
                       PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_data, out_data, numel, shifts_ptr, strides_ptr, sizes_ptr, nums);
  }
};

template <typename DeviceContext, typename T>
class RollGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* out = context.Output<LoDTensor>(framework::GradVarName("X"));
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = framework::stride(input_dim);

    int64_t dim, size;
    size_t gpu_memory_size_ = sizeof(int64_t) * nums;
    std::vector<int64_t> strides, sizes;
    strides.resize(nums);
    sizes.resize(nums);
    paddle::memory::AllocationPtr shifts_gpu =
        memory::Alloc(context.GetPlace(), gpu_memory_size_);
    paddle::memory::AllocationPtr strides_gpu =
        memory::Alloc(context.GetPlace(), gpu_memory_size_);
    paddle::memory::AllocationPtr sizes_gpu =
        memory::Alloc(context.GetPlace(), gpu_memory_size_);

    for (size_t i = 0; i < nums; i++) {
      dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
      size = input_dim[dim];
      shifts[i] = ((0 - shifts[i]) % size + size) % size;
      strides[i] = stride_dim[dim];
      sizes[i] = size;
    }

    paddle::memory::Copy(
        BOOST_GET_CONST(platform::CUDAPlace, shifts_gpu->place()),
        shifts_gpu->ptr(), platform::CPUPlace(), shifts.data(),
        gpu_memory_size_, stream);
    paddle::memory::Copy(
        BOOST_GET_CONST(platform::CUDAPlace, strides_gpu->place()),
        strides_gpu->ptr(), platform::CPUPlace(), strides.data(),
        gpu_memory_size_, stream);
    paddle::memory::Copy(
        BOOST_GET_CONST(platform::CUDAPlace, sizes_gpu->place()),
        sizes_gpu->ptr(), platform::CPUPlace(), sizes.data(), gpu_memory_size_,
        stream);
    int64_t* shifts_ptr = reinterpret_cast<int64_t*>(shifts_gpu->ptr());
    int64_t* strides_ptr = reinterpret_cast<int64_t*>(strides_gpu->ptr());
    int64_t* sizes_ptr = reinterpret_cast<int64_t*>(sizes_gpu->ptr());

    roll_cuda_kernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                           PADDLE_CUDA_NUM_THREADS,
                       PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_data, out_data, numel, shifts_ptr, strides_ptr, sizes_ptr, nums);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roll, ops::RollCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    roll_grad,
    ops::RollGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollGradCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
