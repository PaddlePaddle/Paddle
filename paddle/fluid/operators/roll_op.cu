#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/array.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/roll_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T, size_t N>
__global__ void RollCudaKernel(const T* input, T* output, int64_t Num,
                               paddle::framework::Array<int64_t, N> shifts,
                               paddle::framework::Array<int64_t, N> strides,
                               paddle::framework::Array<int64_t, N> sizes) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= Num) {
    return;
  }

  int64_t output_idx = idx;
  int64_t dim_idx, dim_idx_shift;

#pragma unroll N
  for (size_t i = 0; i < N; i++) {
    dim_idx = (idx / strides[i]) % sizes[i];
    dim_idx_shift = (dim_idx + shifts[i]) % sizes[i];
    output_idx = output_idx + (dim_idx_shift - dim_idx) * strides[i];
  }
  output[output_idx] = input[idx];
}

template <typename T>
class RollKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = framework::stride(input_dim);

    std::vector<int64_t> strides(nums), sizes(nums);
    if (dims.size() == 0) {
      strides[0] = 1;
      sizes[0] = numel;
      shifts[0] = (shifts[0] % numel + numel) % numel;
    } else {
      for (size_t i = 0; i < nums; i++) {
        int dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
        int64_t size = input_dim[dim];

        shifts[i] = (shifts[i] % size + size) % size;
        strides[i] = stride_dim[dim];
        sizes[i] = size;
      }
    }

#define CallRollCudaKernel(N)                                                  \
  case N: {                                                                    \
    paddle::framework::Array<int64_t, N> _strides;                             \
    paddle::framework::Array<int64_t, N> _shifts;                              \
    paddle::framework::Array<int64_t, N> _sizes;                               \
    for (size_t idx = 0; idx < N; ++idx) {                                     \
      _strides[idx] = strides[idx];                                            \
      _shifts[idx] = shifts[idx];                                              \
      _sizes[idx] = sizes[idx];                                                \
    }                                                                          \
    RollCudaKernel<                                                            \
        T,                                                                     \
        N><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,  \
             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_data, out_data, numel,   \
                                                   _shifts, _strides, _sizes); \
    break;                                                                     \
  }

    switch (nums) {
      CallRollCudaKernel(1);
      CallRollCudaKernel(2);
      CallRollCudaKernel(3);
      CallRollCudaKernel(4);
      CallRollCudaKernel(5);
      CallRollCudaKernel(6);
      CallRollCudaKernel(7);
      CallRollCudaKernel(8);
      CallRollCudaKernel(9);
      default:
        break;
    }
  }
};

template <typename T>
class RollGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* out = context.Output<LoDTensor>(framework::GradVarName("X"));
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = framework::stride(input_dim);

    std::vector<int64_t> strides(nums), sizes(nums);
    if (dims.size() == 0) {
      strides[0] = 1;
      sizes[0] = numel;
      shifts[0] = ((-shifts[0]) % numel + numel) % numel;
    } else {
      for (size_t i = 0; i < nums; i++) {
        int dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
        int64_t size = input_dim[dim];

        shifts[i] = ((-shifts[i]) % size + size) % size;
        strides[i] = stride_dim[dim];
        sizes[i] = size;
      }
    }

    switch (nums) {
      CallRollCudaKernel(1);
      CallRollCudaKernel(2);
      CallRollCudaKernel(3);
      CallRollCudaKernel(4);
      CallRollCudaKernel(5);
      CallRollCudaKernel(6);
      CallRollCudaKernel(7);
      CallRollCudaKernel(8);
      CallRollCudaKernel(9);
      default:
        break;
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roll, ops::RollKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    roll_grad, ops::RollGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, int64_t>);
