#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/roll_op.h"
#include "paddle/fluid/platform/complex.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/core/utils/array.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T, size_t Rank>
__global__ void RollCudaKernel(const T* input, T* output, int64_t N,
                               phi::Array<int64_t, Rank> shifts,
                               phi::Array<int64_t, Rank> strides,
                               phi::Array<int64_t, Rank> sizes) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t output_idx = idx;
  int64_t new_dim_idx = 0;

#pragma unroll
  for (size_t i = 0; i < Rank; i++) {
    new_dim_idx = (idx / strides[i]) % sizes[i] + shifts[i];
    if (new_dim_idx >= sizes[i]) {
      output_idx += (shifts[i] - sizes[i]) * strides[i];
    } else {
      output_idx += shifts[i] * strides[i];
    }
  }
  output[output_idx] = input[idx];
}

template <typename T>
class RollKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    if (context.HasInput("ShiftsTensor")) {
      const auto* shifts_tensor =
          context.Input<framework::Tensor>("ShiftsTensor");
      PADDLE_ENFORCE_EQ(
          shifts_tensor->dims().size(), 1,
          platform::errors::InvalidArgument(
              "The rank of ShiftsTensor is expected to be 1, got %s",
              shifts_tensor->dims().size()));
      shifts = GetDataFromTensor<int64_t>(shifts_tensor);
    }
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = phi::stride(input_dim);

    std::vector<int64_t> strides(nums), sizes(nums);
    if (dims.size() == 0) {
      strides[0] = 1;
      sizes[0] = numel;
      shifts[0] = (shifts[0] % numel + numel) % numel;
    } else {
      for (size_t i = 0; i < nums; i++) {
        int dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
        int64_t size = input_dim[dim];

        if (size != 0) {
          shifts[i] = (shifts[i] % size + size) % size;
          strides[i] = stride_dim[dim];
          sizes[i] = size;
        }
      }
    }

#define CALL_ROLL_CUDA_KERNEL(N)                                               \
  case N: {                                                                    \
    phi::Array<int64_t, N> _strides;                                           \
    phi::Array<int64_t, N> _shifts;                                            \
    phi::Array<int64_t, N> _sizes;                                             \
    for (size_t idx = 0; idx < N; ++idx) {                                     \
      _strides[idx] = strides[idx];                                            \
      _shifts[idx] = shifts[idx];                                              \
      _sizes[idx] = sizes[idx];                                                \
    }                                                                          \
    RollCudaKernel<                                                            \
        T,                                                                     \
        N><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,  \
             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_data, out_data, numel,   \
                                                   _shifts, _strides, _sizes); \
    break;                                                                     \
  }

    switch (nums) {
      CALL_ROLL_CUDA_KERNEL(1);
      CALL_ROLL_CUDA_KERNEL(2);
      CALL_ROLL_CUDA_KERNEL(3);
      CALL_ROLL_CUDA_KERNEL(4);
      CALL_ROLL_CUDA_KERNEL(5);
      CALL_ROLL_CUDA_KERNEL(6);
      CALL_ROLL_CUDA_KERNEL(7);
      CALL_ROLL_CUDA_KERNEL(8);
      CALL_ROLL_CUDA_KERNEL(9);
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "shifts.size() should be less than 10, But received shifts.size() "
            "= %d",
            shifts.size()));
    }
  }
};

template <typename T>
class RollGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* out = context.Output<LoDTensor>(framework::GradVarName("X"));
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    if (context.HasInput("ShiftsTensor")) {
      const auto* shifts_tensor =
          context.Input<framework::Tensor>("ShiftsTensor");
      PADDLE_ENFORCE_EQ(
          shifts_tensor->dims().size(), 1,
          platform::errors::InvalidArgument(
              "The rank of ShiftsTensor is expected to be 1, got %s",
              shifts_tensor->dims().size()));
      shifts = GetDataFromTensor<int64_t>(shifts_tensor);
    }
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = phi::stride(input_dim);

    std::vector<int64_t> strides(nums), sizes(nums);
    if (dims.size() == 0) {
      strides[0] = 1;
      sizes[0] = numel;
      shifts[0] = ((-shifts[0]) % numel + numel) % numel;
    } else {
      for (size_t i = 0; i < nums; i++) {
        int dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
        int64_t size = input_dim[dim];
        if (size != 0) {
          shifts[i] = ((-shifts[i]) % size + size) % size;
          strides[i] = stride_dim[dim];
          sizes[i] = size;
        }
      }
    }

    switch (nums) {
      CALL_ROLL_CUDA_KERNEL(1);
      CALL_ROLL_CUDA_KERNEL(2);
      CALL_ROLL_CUDA_KERNEL(3);
      CALL_ROLL_CUDA_KERNEL(4);
      CALL_ROLL_CUDA_KERNEL(5);
      CALL_ROLL_CUDA_KERNEL(6);
      CALL_ROLL_CUDA_KERNEL(7);
      CALL_ROLL_CUDA_KERNEL(8);
      CALL_ROLL_CUDA_KERNEL(9);
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "shifts.size() should be less than 10, But received shifts.size() "
            "= %d",
            shifts.size()));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roll, ops::RollKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::RollKernel<paddle::platform::CUDADeviceContext,
                    paddle::platform::complex<float>>,
    ops::RollKernel<paddle::platform::CUDADeviceContext,
                    paddle::platform::complex<double>>);
REGISTER_OP_CUDA_KERNEL(
    roll_grad, ops::RollGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext,
                        paddle::platform::complex<float>>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext,
                        paddle::platform::complex<double>>);
