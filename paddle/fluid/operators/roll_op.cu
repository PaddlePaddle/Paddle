#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/array.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/roll_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T, size_t Rank>
__global__ void RollCudaKernel(const T* input, T* output, int64_t N,
                               paddle::framework::Array<int64_t, Rank> shifts,
                               paddle::framework::Array<int64_t, Rank> strides,
                               paddle::framework::Array<int64_t, Rank> sizes) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t output_idx = idx;
  int64_t dim_idx, dim_idx_shift;

#pragma unroll Rank
  for (size_t i = 0; i < Rank; i++) {
    dim_idx = (idx / strides[i]) % sizes[i];
    dim_idx_shift = (dim_idx + shifts[i]) % sizes[i];
    output_idx = output_idx + (dim_idx_shift - dim_idx) * strides[i];
  }
  output[output_idx] = input[idx];
}

template <typename T>
class RollKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = framework::stride(input_dim);

    std::vector<int64_t> strides(nums), sizes(nums);
    if (dims.size() == 0) {
      strides[0] = 1;
      sizes[0] = numel;
      shifts[0] = (shifts[0] % numel + numel) % numel;
    } else {
      for (size_t i = 0; i < nums; i++) {
        int dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
        int64_t size = input_dim[dim];

        shifts[i] = (shifts[i] % size + size) % size;
        strides[i] = stride_dim[dim];
        sizes[i] = size;
      }
    }

#define CALL_ROLL_CUDA_KERNEL(N)                                               \
  case N: {                                                                    \
    paddle::framework::Array<int64_t, N> _strides;                             \
    paddle::framework::Array<int64_t, N> _shifts;                              \
    paddle::framework::Array<int64_t, N> _sizes;                               \
    for (size_t idx = 0; idx < N; ++idx) {                                     \
      _strides[idx] = strides[idx];                                            \
      _shifts[idx] = shifts[idx];                                              \
      _sizes[idx] = sizes[idx];                                                \
    }                                                                          \
    RollCudaKernel<                                                            \
        T,                                                                     \
        N><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,  \
             PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_data, out_data, numel,   \
                                                   _shifts, _strides, _sizes); \
    break;                                                                     \
  }

    switch (nums) {
      CALL_ROLL_CUDA_KERNEL(1);
      CALL_ROLL_CUDA_KERNEL(2);
      CALL_ROLL_CUDA_KERNEL(3);
      CALL_ROLL_CUDA_KERNEL(4);
      CALL_ROLL_CUDA_KERNEL(5);
      CALL_ROLL_CUDA_KERNEL(6);
      CALL_ROLL_CUDA_KERNEL(7);
      CALL_ROLL_CUDA_KERNEL(8);
      CALL_ROLL_CUDA_KERNEL(9);
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "shifts.size() should be less than 10, But received shifts.size() "
            "= %d",
            shifts.size()));
    }
  }
};

template <typename T>
class RollGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>(framework::GradVarName("Out"));
    auto* out = context.Output<LoDTensor>(framework::GradVarName("X"));
    std::vector<int64_t> shifts = context.Attr<std::vector<int64_t>>("shifts");
    std::vector<int64_t> dims = context.Attr<std::vector<int64_t>>("axis");

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = in->numel();
    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();
    size_t nums = shifts.size();
    auto input_dim = in->dims();
    auto stride_dim = framework::stride(input_dim);

    std::vector<int64_t> strides(nums), sizes(nums);
    if (dims.size() == 0) {
      strides[0] = 1;
      sizes[0] = numel;
      shifts[0] = ((-shifts[0]) % numel + numel) % numel;
    } else {
      for (size_t i = 0; i < nums; i++) {
        int dim = dims[i] >= 0 ? dims[i] : dims[i] + input_dim.size();
        int64_t size = input_dim[dim];

        shifts[i] = ((-shifts[i]) % size + size) % size;
        strides[i] = stride_dim[dim];
        sizes[i] = size;
      }
    }

    switch (nums) {
      CALL_ROLL_CUDA_KERNEL(1);
      CALL_ROLL_CUDA_KERNEL(2);
      CALL_ROLL_CUDA_KERNEL(3);
      CALL_ROLL_CUDA_KERNEL(4);
      CALL_ROLL_CUDA_KERNEL(5);
      CALL_ROLL_CUDA_KERNEL(6);
      CALL_ROLL_CUDA_KERNEL(7);
      CALL_ROLL_CUDA_KERNEL(8);
      CALL_ROLL_CUDA_KERNEL(9);
      default:
        PADDLE_THROW(platform::errors::InvalidArgument(
            "shifts.size() should be less than 10, But received shifts.size() "
            "= %d",
            shifts.size()));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    roll, ops::RollKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    roll_grad, ops::RollGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, double>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::RollGradKernel<paddle::platform::CUDADeviceContext, int64_t>);
