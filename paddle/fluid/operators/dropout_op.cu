/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/dropout_impl.cu.h"
#include "paddle/fluid/operators/dropout_op.h"
#include "paddle/fluid/platform/bfloat16.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

// It seems that Eigen::Tensor::setRandom in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename Place, typename T>
class GPUDropoutKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* seed =
        context.HasInput("Seed") ? context.Input<Tensor>("Seed") : nullptr;
    auto* y = context.Output<Tensor>("Out");
    y->mutable_data<T>(context.GetPlace());
    float dropout_prob = context.Attr<float>("dropout_prob");

    auto& dropout_implementation =
        context.Attr<std::string>("dropout_implementation");
    bool upscale_in_train = (dropout_implementation == "upscale_in_train");

    bool is_test = context.Attr<bool>("is_test");

    auto& dev_ctx = context.cuda_device_context();
    auto* mask = context.Output<Tensor>("Mask");
    mask->mutable_data<uint8_t>(context.GetPlace());

    bool is_fix_seed = context.Attr<bool>("fix_seed");
    int seed_val = context.Attr<int>("seed");
    DropoutFwGPUKernelDriver<T>(dev_ctx, is_test, dropout_implementation,
                                dropout_prob, upscale_in_train, is_fix_seed,
                                seed_val, *x, seed, mask, y);
  }
};

template <typename DeviceContext, typename T>
class GPUDropoutGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* grad_x = context.Output<Tensor>(framework::GradVarName("X"));
    auto* grad_y = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* mask = context.Input<Tensor>("Mask");
    grad_x->mutable_data<T>(context.GetPlace());
    auto size = grad_x->numel();
    auto& dropout_implementation =
        context.Attr<std::string>("dropout_implementation");
    float dropout_prob = context.Attr<float>("dropout_prob");

    bool is_test = context.Attr<bool>("is_test");

    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    DropoutGradGPUKernelDriver<T>(dev_ctx, dropout_implementation, dropout_prob,
                                  *grad_y, *mask, size, grad_x, is_test);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    dropout, ops::GPUDropoutKernel<plat::CUDADeviceContext, float>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, plat::float16>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, plat::bfloat16>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    dropout_grad, ops::GPUDropoutGradKernel<plat::CUDADeviceContext, float>,
    ops::GPUDropoutGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::GPUDropoutGradKernel<plat::CUDADeviceContext, plat::bfloat16>,
    ops::GPUDropoutGradKernel<plat::CUDADeviceContext, double>);
