#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "paddle/fluid/platform/dynload/hiprand.h"
#endif
#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
#include "paddle/fluid/platform/dynload/hiprand.h"
#endif
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <algorithm>
#include <string>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/dropout_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T, typename MaskType>
__global__ void RandomGenerator(const size_t n, uint64_t seed,
                                const float dropout_prob, const T* src,
                                MaskType* mask, T* dst,
                                bool is_upscale_in_train, uint64_t increment) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
#ifdef PADDLE_WITH_HIP
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, increment, &state);
#else
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, increment, &state);
#endif

  MaskType mask_val;
  T dst_val;
  T factor = static_cast<T>(1.0f / (1.0f - dropout_prob));
  for (; idx < n; idx += blockDim.x * gridDim.x) {
    T src_val = src[idx];
#ifdef PADDLE_WITH_HIP
    if (hiprand_uniform(&state) < dropout_prob) {
#else
    if (hiprand_uniform(&state) < dropout_prob) {
#endif
      mask_val = 0;
      dst_val = 0;
    } else {
      mask_val = 1;
      dst_val = is_upscale_in_train ? src_val * factor : src_val;
    }
    mask[idx] = mask_val;
    dst[idx] = dst_val;
  }
}

template <typename T, typename MaskType, int VecSize>
__global__ void VectorizedRandomGenerator(const size_t n, uint64_t seed,
                                          const float dropout_prob,
                                          const T* src, MaskType* mask, T* dst,
                                          bool is_upscale_in_train,
                                          uint64_t increment) {
  using LoadT = platform::AlignedVector<T, VecSize>;
  using MaskLoadT = platform::AlignedVector<MaskType, VecSize>;

#ifdef PADDLE_WITH_HIP
  int64_t idx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, increment, &state);
#else
  int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, increment, &state);
#endif

  T factor = static_cast<T>(1.0f / (1.0f - dropout_prob));
  for (int i = idx * VecSize; i < n; i += blockDim.x * gridDim.x * VecSize) {
    LoadT src_val;
    platform::Load<T, VecSize>(&src[i], &src_val);

#ifdef PADDLE_WITH_HIP
    float4 rand = hiprand_uniform4(&state);
#else
    float4 rand = hiprand_uniform4(&state);
#endif

    LoadT dst_val;
    MaskLoadT mask_val;

#pragma unroll
    for (int j = 0; j < VecSize; j++) {
      if ((&rand.x)[j] < dropout_prob) {
        dst_val[j] = 0;
        mask_val[j] = 0;
      } else {
        dst_val[j] = is_upscale_in_train ? src_val[j] * factor : src_val[j];
        mask_val[j] = 1;
      }
    }

    platform::Store<T, VecSize>(dst_val, &dst[i]);
    platform::Store<MaskType, VecSize>(mask_val, &mask[i]);
  }
}

// It seems that Eigen::Tensor::setRandom in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename Place, typename T>
class GPUDropoutKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* seed =
        context.HasInput("Seed") ? context.Input<Tensor>("Seed") : nullptr;
    auto* y = context.Output<Tensor>("Out");
    y->mutable_data<T>(context.GetPlace());
    float dropout_prob = context.Attr<float>("dropout_prob");

    auto& dropout_implementation =
        context.Attr<std::string>("dropout_implementation");
    bool upscale_in_train = (dropout_implementation == "upscale_in_train");

    auto& place = *context.template device_context<Place>().eigen_device();
    if (!context.Attr<bool>("is_test")) {
      int64_t x_numel = x->numel();
      auto stream = context.cuda_device_context().stream();

      auto* mask = context.Output<Tensor>("Mask");
      auto* mask_data = mask->mutable_data<uint8_t>(context.GetPlace());
      size_t size = framework::product(mask->dims());
      auto* x_data = x->data<T>();
      auto* y_data = y->mutable_data<T>(context.GetPlace());
      if (dropout_prob == 1.0f) {
#ifdef PADDLE_WITH_HIP
        PADDLE_ENFORCE_CUDA_SUCCESS(
            hipMemsetAsync(y_data, 0, x_numel * sizeof(T), stream));
        PADDLE_ENFORCE_CUDA_SUCCESS(
            hipMemsetAsync(mask_data, 0, x_numel * sizeof(*mask_data), stream));
#else
        PADDLE_ENFORCE_CUDA_SUCCESS(
            hipMemsetAsync(y_data, 0, x_numel * sizeof(T), stream));
        PADDLE_ENFORCE_CUDA_SUCCESS(hipMemsetAsync(
            mask_data, 0, x_numel * sizeof(*mask_data), stream));
#endif
        return;
      }

      const auto& dev_ctx = context.cuda_device_context();
      platform::GpuLaunchConfig config =
          platform::GetGpuLaunchConfig1D(dev_ctx, size);

      // increment is used to set the args(offset) of hiprand_init, which defines
      // offset in subsequence.
      // The detail:
      // https://docs.nvidia.com/cuda/hiprand/device-api-overview.html
      // Increment should be at least the number of hiprand() random numbers used
      // in each thread to avoid the random number generated this time being the
      // same as the previous calls.
      uint64_t seed_data;
      uint64_t increment;
      int vec_size = platform::GetVectorizedSize<T>(x_data);
      auto offset = ((x_numel - 1) / (config.block_per_grid.x *
                                      config.thread_per_block.x * vec_size) +
                     1) *
                    vec_size;
      int device_id = BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace())
                          .GetDeviceId();
      auto gen_cuda = framework::GetDefaultCUDAGenerator(device_id);

      if (seed && platform::is_gpu_place(seed->place())) {
        framework::Tensor seed_cpu_tensor;
        TensorCopySync(*seed, platform::CPUPlace(), &seed_cpu_tensor);
        seed_data = static_cast<uint64_t>(seed_cpu_tensor.data<int>()[0]);
        increment = offset;
      } else if (gen_cuda->GetIsInitPy() && (!context.Attr<bool>("fix_seed"))) {
        auto seed_offset = gen_cuda->IncrementOffset(offset);
        seed_data = seed_offset.first;
        increment = seed_offset.second;
      } else {
        if (seed) {
          seed_data = *(seed->data<int>());
        } else {
          std::random_device rnd;
          seed_data = context.Attr<bool>("fix_seed") ? context.Attr<int>("seed")
                                                     : rnd();
        }
        increment = offset;
      }

#ifdef __HIPCC__
      if (vec_size == 4 && size % 4 == 0) {
        hipLaunchKernelGGL(
            HIP_KERNEL_NAME(VectorizedRandomGenerator<T, uint8_t, 4>),
            config.block_per_grid, config.thread_per_block, 0, stream, size,
            seed_data, dropout_prob, x_data, mask_data, y_data,
            upscale_in_train, increment);
      } else {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(RandomGenerator<T, uint8_t>),
                           config.block_per_grid, config.thread_per_block, 0,
                           stream, size, seed_data, dropout_prob, x_data,
                           mask_data, y_data, upscale_in_train, increment);
      }
#else
      if (vec_size == 4 && size % 4 == 0) {
        VectorizedRandomGenerator<
            T, uint8_t,
            4><<<config.block_per_grid, config.thread_per_block, 0, stream>>>(
            size, seed_data, dropout_prob, x_data, mask_data, y_data,
            upscale_in_train, increment);
      } else {
        RandomGenerator<T, uint8_t><<<config.block_per_grid,
                                      config.thread_per_block, 0, stream>>>(
            size, seed_data, dropout_prob, x_data, mask_data, y_data,
            upscale_in_train, increment);
      }
#endif
    } else {
      auto X = EigenMatrix<T>::Reshape(*x, 1);
      auto Y = EigenMatrix<T>::Reshape(*y, 1);
      if (upscale_in_train) {
        Y.device(place) = X;
      } else {
        Y.device(place) = X * static_cast<T>(1.0f - dropout_prob);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    dropout, ops::GPUDropoutKernel<plat::CUDADeviceContext, float>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, plat::float16>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    dropout_grad, ops::DropoutGradKernel<plat::CUDADeviceContext, float>,
    ops::DropoutGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::DropoutGradKernel<plat::CUDADeviceContext, double>);
