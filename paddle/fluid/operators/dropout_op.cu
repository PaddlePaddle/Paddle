#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <string>
#include "paddle/fluid/operators/dropout_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T, typename MaskType>
__global__ void RandomGenerator(const size_t n, const int seed,
                                const float dropout_prob, const T* src,
                                MaskType* mask_data, T* dst,
                                bool is_upscale_in_train) {
  thrust::minstd_rand rng;
  rng.seed(seed);
  thrust::uniform_real_distribution<float> dist(0, 1);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = 0;

  MaskType mask;
  T dest;
  for (; idx < n; idx += blockDim.x * gridDim.x) {
    T s = src[idx];
    if (step_size == 0) {
      rng.discard(idx);
      step_size = blockDim.x * gridDim.x;
    } else {
      rng.discard(step_size);
    }
    if (dist(rng) < dropout_prob) {
      mask = 0;
      dest = 0;
    } else {
      mask = 1;
      if (is_upscale_in_train) {
        dest = s / static_cast<T>(1.0f - dropout_prob);
      } else {
        dest = s;
      }
    }
    mask_data[idx] = mask;
    dst[idx] = dest;
  }
}

// It seems that Eigen::Tensor::setRandom in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename Place, typename T>
class GPUDropoutKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* y = context.Output<Tensor>("Out");
    y->mutable_data<T>(context.GetPlace());
    float dropout_prob = context.Attr<float>("dropout_prob");

    auto& dropout_implementation =
        context.Attr<std::string>("dropout_implementation");
    bool upscale_in_train = (dropout_implementation == "upscale_in_train");

    auto& place = *context.template device_context<Place>().eigen_device();
    if (!context.Attr<bool>("is_test")) {
      int64_t x_numel = x->numel();
      auto stream = context.cuda_device_context().stream();

      auto* mask = context.Output<Tensor>("Mask");
      auto* mask_data = mask->mutable_data<uint8_t>(context.GetPlace());
      size_t size = framework::product(mask->dims());
      auto* x_data = x->data<T>();
      auto* y_data = y->mutable_data<T>(context.GetPlace());
      if (dropout_prob == 1.0f) {
        PADDLE_ENFORCE(hipMemsetAsync(y_data, 0, x_numel * sizeof(T), stream));
        PADDLE_ENFORCE(hipMemsetAsync(mask_data, 0,
                                       x_numel * sizeof(*mask_data), stream));
        return;
      }

      std::random_device rnd;
      int seed =
          context.Attr<bool>("fix_seed") ? context.Attr<int>("seed") : rnd();

      int threads = 512;
      int grid = (x_numel + threads - 1) / threads;
      RandomGenerator<T, uint8_t><<<grid, threads, 0, stream>>>(
          size, seed, dropout_prob, x_data, mask_data, y_data,
          upscale_in_train);
    } else {
      auto X = EigenMatrix<T>::Reshape(*x, 1);
      auto Y = EigenMatrix<T>::Reshape(*y, 1);
      if (upscale_in_train) {
        Y.device(place) = X;
      } else {
        Y.device(place) = X * static_cast<T>(1.0f - dropout_prob);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    dropout, ops::GPUDropoutKernel<plat::CUDADeviceContext, float>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, plat::float16>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    dropout_grad, ops::DropoutGradKernel<plat::CUDADeviceContext, float>,
    ops::DropoutGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::DropoutGradKernel<plat::CUDADeviceContext, double>);
