#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/operators/dropout_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void RandomGenerator(const size_t n, const int seed,
                                const float dropout_prob, const T* src,
                                T* mask_data, T* dst) {
  thrust::minstd_rand rng;
  rng.seed(seed);
  thrust::uniform_real_distribution<float> dist(0, 1);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int step_size = 0;

  T mask;
  T dest;
  for (; idx < n; idx += blockDim.x * gridDim.x) {
    T s = src[idx];
    if (step_size == 0) {
      rng.discard(idx);
      step_size = blockDim.x * gridDim.x;
    } else {
      rng.discard(step_size);
    }
    if (dist(rng) < dropout_prob) {
      mask = static_cast<T>(0);
    } else {
      mask = static_cast<T>(1);
    }
    dest = s * mask;
    mask_data[idx] = mask;
    dst[idx] = dest;
  }
}

// It seems that Eigen::Tensor::setRandom in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename Place, typename T>
class GPUDropoutKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* y = context.Output<Tensor>("Out");
    y->mutable_data<T>(context.GetPlace());
    float dropout_prob = context.Attr<float>("dropout_prob");

    auto& place = *context.template device_context<Place>().eigen_device();
    if (!context.Attr<bool>("is_test")) {
      auto* mask = context.Output<Tensor>("Mask");
      auto* mask_data = mask->mutable_data<T>(context.GetPlace());
      size_t size = framework::product(mask->dims());
      auto* x_data = x->data<T>();
      auto* y_data = y->mutable_data<T>(context.GetPlace());

      std::random_device rnd;
      int seed =
          context.Attr<bool>("fix_seed") ? context.Attr<int>("seed") : rnd();

      int threads = 512;
      int grid = (x->numel() + threads - 1) / threads;
      RandomGenerator<
          T><<<grid, threads, 0, context.cuda_device_context().stream()>>>(
          size, seed, dropout_prob, x_data, mask_data, y_data);
    } else {
      auto X = EigenMatrix<T>::Reshape(*x, 1);
      auto Y = EigenMatrix<T>::Reshape(*y, 1);
      Y.device(place) = X * static_cast<T>(1.0f - dropout_prob);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    dropout, ops::GPUDropoutKernel<plat::CUDADeviceContext, float>,
    ops::GPUDropoutKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(dropout_grad,
                        ops::DropoutGradKernel<plat::CUDADeviceContext, float>);
