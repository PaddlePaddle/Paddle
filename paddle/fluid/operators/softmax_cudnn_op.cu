#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/operators/softmax_op.h"
#include "paddle/fluid/platform/cudnn_helper.h"

namespace paddle {
namespace platform {
struct CUDAPlace;
struct float16;
}  // namespace platform
}  // namespace paddle

namespace paddle {
namespace operators {

using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
using Tensor = framework::Tensor;

static inline int SizeOutAxis(const int axis, DDim dims) {
  int size = 1;
  for (int i = axis + 1; i < dims.size(); i++) {
    size *= dims[i];
  }
  return size;
}

template <typename T, typename VECT, int VPT, int WARP_PER_BLOCK>
__global__ void VecSoftmaxKernel(T* dst, const T* src, const int batch_size,
                                 const int softmax_ele) {
  int offset = blockIdx.x * softmax_ele * WARP_PER_BLOCK;
  int idx = threadIdx.x * VPT;

  VECT buf = reinterpret_cast<const VECT*>(&src[offset + idx])[0];
  T* bufp = reinterpret_cast<T*>(&buf);
  float4 val4;
  float* val4p = reinterpret_cast<float*>(&val4);
  for (int i = 0; i < VPT; ++i) {
    val4p[i] = static_cast<float>(bufp[i]);
  }
  float val = val4.x + val4.y + val4.z + val4.w;
  float max_val = math::warpReduceMax<float>(
      max(max(val4.x, val4.y), max(val4.z, val4.w)), 0xffffffff);
  float4 tmp4 = make_float4(__expf(val4.x - max_val), __expf(val4.y - max_val),
                            __expf(val4.z - max_val), __expf(val4.w - max_val));
  float* tmp4p = reinterpret_cast<float*>(&tmp4);
  float invsum = 1.f / (math::warpReduceSum<float>(
                            tmp4.x + tmp4.y + tmp4.z + tmp4.w, 0xffffffff) +
                        1e-6f);
  for (int i = 0; i < VPT; ++i) {
    bufp[i] = static_cast<T>(tmp4p[i] * invsum);
  }
  reinterpret_cast<VECT*>(&dst[offset + idx])[0] = buf;
}

template <typename T>
class SoftmaxCUDNNKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* out = ctx.Output<Tensor>("Out");
    out->mutable_data<T>(ctx.GetPlace());
    auto* out_data = out->data<T>();

    auto dims = x->dims();
    const int rank = dims.size();
    const int axis = CanonicalAxis(ctx.Attr<int>("axis"), rank);
    const int dim = dims[axis];
    const int N = SizeToAxis(axis, dims);
    const int D = SizeOutAxis(axis, dims);

    if (D == 1 && dim == 128 && N % 4 == 0 && sizeof(T) <= 4) {
      // a warp for a batch, 4 elements for a thread, only support the softmax
      // dim size = 128 currently
      constexpr int warps_per_block = 4;
      if (sizeof(T) == 2) {
        VecSoftmaxKernel<
            T, int2, 4,
            warps_per_block><<<N / warps_per_block, warps_per_block * WARP_SIZE,
                               0, ctx.cuda_device_context().stream()>>>(
            out_data, x->data<T>(), N, dim);
      } else if (sizeof(T) == 4) {
        VecSoftmaxKernel<
            T, int4, 4,
            warps_per_block><<<N / warps_per_block, warps_per_block * WARP_SIZE,
                               0, ctx.cuda_device_context().stream()>>>(
            out_data, x->data<T>(), N, dim);
      } else {
        assert(false && "not support");
      }
    } else {
      ScopedTensorDescriptor desc;
      std::vector<int> tensor_dims = {N, dim, D, 1};
      DataLayout layout = DataLayout::kNCHW;
      hipdnnTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);

      auto& dev_ctx =
          ctx.template device_context<platform::CUDADeviceContext>();
      auto handle = dev_ctx.cudnn_handle();
      auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                   : HIPDNN_SOFTMAX_MODE_CHANNEL;

      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
          handle, HIPDNN_SOFTMAX_ACCURATE, mode,
          platform::CudnnDataType<T>::kOne(), desc_, x->data<T>(),
          platform::CudnnDataType<T>::kZero(), desc_, out_data));
    }
  }
};

template <typename T>
class SoftmaxGradCUDNNKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* out = ctx.Input<Tensor>("Out");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    dx->mutable_data<T>(ctx.GetPlace());
    auto* dx_data = dx->data<T>();

    auto dims = out->dims();
    const int rank = dims.size();
    const int axis = CanonicalAxis(ctx.Attr<int>("axis"), rank);
    const int dim = dims[axis];
    const int N = SizeToAxis(axis, dims);
    const int D = SizeOutAxis(axis, dims);

    ScopedTensorDescriptor desc;
    std::vector<int> tensor_dims = {N, dim, D, 1};
    DataLayout layout = DataLayout::kNCHW;
    hipdnnTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);

    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto handle = dev_ctx.cudnn_handle();
    auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                 : HIPDNN_SOFTMAX_MODE_CHANNEL;

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxBackward(
        handle, HIPDNN_SOFTMAX_ACCURATE, mode,
        platform::CudnnDataType<T>::kOne(), desc_, out->data<T>(), desc_,
        dout->data<T>(), platform::CudnnDataType<T>::kZero(), desc_, dx_data));
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_KERNEL(softmax, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxCUDNNKernel<float>,
                   ops::SoftmaxCUDNNKernel<double>,
                   ops::SoftmaxCUDNNKernel<plat::float16>);
REGISTER_OP_KERNEL(softmax_grad, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxGradCUDNNKernel<float>,
                   ops::SoftmaxGradCUDNNKernel<double>,
                   ops::SoftmaxGradCUDNNKernel<plat::float16>);
