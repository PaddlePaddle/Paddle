#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <functional>
#include <list>
#include <memory>
#include <mutex>
#include <numeric>
#include <sstream>
#include <stdexcept>
#include <string>
#include <unordered_map>
#include <vector>

#include "paddle/fluid/operators/conj_op.h"
#include "paddle/fluid/operators/spectral_op.h"
#include "paddle/fluid/operators/transpose_op.h"

namespace paddle {
namespace operators {

namespace {

using ScalarType = framework::proto::VarType::Type;
const int64_t kMaxCUFFTNdim = 3;
const int64_t kMaxDataNdim = kMaxCUFFTNdim + 1;

std::ostream& operator<<(std::ostream& os, FFTTransformType fft_type) {
  std::string repr;
  switch (fft_type) {
    case FFTTransformType::C2C:
      repr = "C2C";
      break;
    case FFTTransformType::C2R:
      repr = "C2R";
      break;
    case FFTTransformType::R2C:
      repr = "R2C";
      break;
    default:
      repr = "UNK";
  }
  os << repr;
  return os;
}

static inline std::string get_cufft_error_info(hipfftResult error) {
  switch (error) {
    case HIPFFT_SUCCESS:
      return "HIPFFT_SUCCESS";
    case HIPFFT_INVALID_PLAN:
      return "HIPFFT_INVALID_PLAN";
    case HIPFFT_ALLOC_FAILED:
      return "HIPFFT_ALLOC_FAILED";
    case HIPFFT_INVALID_TYPE:
      return "HIPFFT_INVALID_TYPE";
    case HIPFFT_INVALID_VALUE:
      return "HIPFFT_INVALID_VALUE";
    case HIPFFT_INTERNAL_ERROR:
      return "HIPFFT_INTERNAL_ERROR";
    case HIPFFT_EXEC_FAILED:
      return "HIPFFT_EXEC_FAILED";
    case HIPFFT_SETUP_FAILED:
      return "HIPFFT_SETUP_FAILED";
    case HIPFFT_INVALID_SIZE:
      return "HIPFFT_INVALID_SIZE";
    case HIPFFT_UNALIGNED_DATA:
      return "HIPFFT_UNALIGNED_DATA";
    case HIPFFT_INCOMPLETE_PARAMETER_LIST:
      return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
    case HIPFFT_INVALID_DEVICE:
      return "HIPFFT_INVALID_DEVICE";
    case HIPFFT_PARSE_ERROR:
      return "HIPFFT_PARSE_ERROR";
    case HIPFFT_NO_WORKSPACE:
      return "HIPFFT_NO_WORKSPACE";
    case HIPFFT_NOT_IMPLEMENTED:
      return "HIPFFT_NOT_IMPLEMENTED";
#ifndef __HIPCC__
    case CUFFT_LICENSE_ERROR:
      return "CUFFT_LICENSE_ERROR";
#endif
    case HIPFFT_NOT_SUPPORTED:
      return "HIPFFT_NOT_SUPPORTED";
    default:
      std::ostringstream ss;
      ss << "unknown error " << error;
      return ss.str();
  }
}

static inline void CUFFT_CHECK(hipfftResult error) {
  if (error != HIPFFT_SUCCESS) {
    PADDLE_THROW(platform::errors::External(get_cufft_error_info(error)));
  }
}

// This struct is used to easily compute hashes of the
// parameters. It will be the **key** to the plan cache.
struct PlanKey {
  // between 1 and kMaxCUFFTNdim, i.e., 1 <= signal_ndim <= 3
  int64_t signal_ndim_;
  // These include additional batch dimension as well.
  int64_t sizes_[kMaxDataNdim];
  int64_t input_shape_[kMaxDataNdim];
  int64_t output_shape_[kMaxDataNdim];
  FFTTransformType fft_type_;
  ScalarType value_type_;

  PlanKey() = default;

  PlanKey(const std::vector<int64_t>& in_shape,
          const std::vector<int64_t>& out_shape,
          const std::vector<int64_t>& signal_size, FFTTransformType fft_type,
          ScalarType value_type) {
    // Padding bits must be zeroed for hashing
    memset(this, 0, sizeof(*this));
    signal_ndim_ = signal_size.size() - 1;
    fft_type_ = fft_type;
    value_type_ = value_type;

    std::copy(signal_size.cbegin(), signal_size.cend(), sizes_);
    std::copy(in_shape.cbegin(), in_shape.cend(), input_shape_);
    std::copy(out_shape.cbegin(), out_shape.cend(), output_shape_);
  }
};

// An RAII encapsulation of cuFFTHandle
class CuFFTHandle {
  ::hipfftHandle handle_;

 public:
  CuFFTHandle() { CUFFT_CHECK(hipfftCreate(&handle_)); }

  ::hipfftHandle& get() { return handle_; }
  const ::hipfftHandle& get() const { return handle_; }

  ~CuFFTHandle() {
// Not using fftDestroy() for rocFFT to work around double freeing of handles
#ifndef __HIPCC__
    hipfftDestroy(handle_);
#endif
  }
};

#ifdef __HIPCC__
using plan_size_type = int;
#else
using plan_size_type = long long int;  // NOLINT
#endif

// This class contains all the information needed to execute a cuFFT plan:
//   1. the plan
//   //2. whether to clone input before executing the plan
//   2. the workspace size needed
//
// This class will be the **value** in the plan cache.
// It **owns** the raw plan via a unique_ptr.
class CuFFTConfig {
 public:
  // Only move semantics is enought for this class. Although we already use
  // unique_ptr for the plan, still remove copy constructor and assignment op so
  // we don't accidentally copy and take perf hit.
  CuFFTConfig(const CuFFTConfig&) = delete;
  CuFFTConfig& operator=(CuFFTConfig const&) = delete;

  explicit CuFFTConfig(const PlanKey& params)
      : CuFFTConfig(std::vector<int64_t>(
                        params.sizes_, params.sizes_ + params.signal_ndim_ + 1),
                    params.signal_ndim_, params.fft_type_, params.value_type_) {
  }

  // sizes are full signal, including batch size and always two-sided
  CuFFTConfig(std::vector<int64_t> sizes, const int64_t signal_ndim,
              FFTTransformType fft_type, ScalarType dtype)
      : fft_type_(fft_type), value_type_(dtype) {
    // signal sizes (excluding batch dim)
    std::vector<plan_size_type> signal_sizes(sizes.begin() + 1, sizes.end());

    // input batch size
    const auto batch = static_cast<plan_size_type>(sizes[0]);
    // const int64_t signal_ndim = sizes.size() - 1;
    PADDLE_ENFORCE_EQ(signal_ndim, sizes.size() - 1,
                      platform::errors::InvalidArgument(
                          "The signal_ndim must be equal to sizes.size() - 1,"
                          "But signal_ndim is: [%d], sizes.size() - 1 is: [%d]",
                          signal_ndim, sizes.size() - 1));

#ifdef __HIPCC__
    hipfftType exec_type = [&] {
      if (dtype == framework::proto::VarType::FP32) {
        switch (fft_type) {
          case FFTTransformType::C2C:
            return HIPFFT_C2C;
          case FFTTransformType::R2C:
            return HIPFFT_R2C;
          case FFTTransformType::C2R:
            return HIPFFT_C2R;
        }
      } else if (dtype == framework::proto::VarType::FP64) {
        switch (fft_type) {
          case FFTTransformType::C2C:
            return HIPFFT_Z2Z;
          case FFTTransformType::R2C:
            return HIPFFT_D2Z;
          case FFTTransformType::C2R:
            return HIPFFT_Z2D;
        }
      }
      PADDLE_THROW(platform::errors::InvalidArgument(
          "hipFFT only support transforms of type float32 and float64"));
    }();
#else
    hipDataType itype, otype, exec_type;
    const auto complex_input = has_complex_input(fft_type);
    const auto complex_output = has_complex_output(fft_type);
    if (dtype == framework::proto::VarType::FP32) {
      itype = complex_input ? HIP_C_32F : HIP_R_32F;
      otype = complex_output ? HIP_C_32F : HIP_R_32F;
      exec_type = HIP_C_32F;
    } else if (dtype == framework::proto::VarType::FP64) {
      itype = complex_input ? HIP_C_64F : HIP_R_64F;
      otype = complex_output ? HIP_C_64F : HIP_R_64F;
      exec_type = HIP_C_64F;
    } else if (dtype == framework::proto::VarType::FP16) {
      itype = complex_input ? HIP_C_16F : HIP_R_16F;
      otype = complex_output ? HIP_C_16F : HIP_R_16F;
      exec_type = HIP_C_16F;
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "cuFFT only support transforms of type float16, float32 and "
          "float64"));
    }
#endif

    // disable auto allocation of workspace to use THC allocator
    CUFFT_CHECK(hipfftSetAutoAllocation(plan(), /* autoAllocate */ 0));

    size_t ws_size_t;

// make plan
#ifdef __HIPCC__
    CUFFT_CHECK(hipfftMakePlanMany(
        plan(), signal_ndim, signal_sizes.data(),
        /* inembed */ nullptr, /* base_istride */ 1, /* idist */ 1,
        /* onembed */ nullptr, /* base_ostride */ 1, /* odist */ 1, exec_type,
        batch, &ws_size_t));
#else

    CUFFT_CHECK(hipfftXtMakePlanMany(
        plan(), signal_ndim, signal_sizes.data(),
        /* inembed */ nullptr, /* base_istride */ 1, /* idist */ 1, itype,
        /* onembed */ nullptr, /* base_ostride */ 1, /* odist */ 1, otype,
        batch, &ws_size_t, exec_type));
#endif

    ws_size = ws_size_t;
  }

  const hipfftHandle& plan() const { return plan_ptr.get(); }

  FFTTransformType transform_type() const { return fft_type_; }
  ScalarType data_type() const { return value_type_; }
  size_t workspace_size() const { return ws_size; }

 private:
  CuFFTHandle plan_ptr;
  size_t ws_size;
  FFTTransformType fft_type_;
  ScalarType value_type_;
};

// Hashing machinery for Key
// Fowler–Noll–Vo hash function
// see
// https://en.wikipedia.org/wiki/Fowler%E2%80%93Noll%E2%80%93Vo_hash_function
template <typename Key>
struct KeyHash {
  // Key must be a POD because we read out its memory
  // contenst as char* when hashing
  static_assert(std::is_pod<Key>::value, "Key must be plain old data type");

  size_t operator()(const Key& params) const {
    auto ptr = reinterpret_cast<const uint8_t*>(&params);
    uint32_t value = 0x811C9DC5;
    for (int i = 0; i < static_cast<int>(sizeof(Key)); ++i) {
      value ^= ptr[i];
      value *= 0x01000193;
    }
    return static_cast<size_t>(value);
  }
};

template <typename Key>
struct KeyEqual {
  // Key must be a POD because we read out its memory
  // contenst as char* when comparing
  static_assert(std::is_pod<Key>::value, "Key must be plain old data type");

  bool operator()(const Key& a, const Key& b) const {
    auto ptr1 = reinterpret_cast<const uint8_t*>(&a);
    auto ptr2 = reinterpret_cast<const uint8_t*>(&b);
    return memcmp(ptr1, ptr2, sizeof(Key)) == 0;
  }
};

#if CUDA_VERSION < 10000
// Note that the max plan number for CUDA version < 10 has to be 1023
// due to a bug that fails on the 1024th plan
constexpr size_t CUFFT_MAX_PLAN_NUM = 1023;
constexpr size_t CUFFT_DEFAULT_CACHE_SIZE = CUFFT_MAX_PLAN_NUM;
#else
constexpr size_t CUFFT_MAX_PLAN_NUM = std::numeric_limits<size_t>::max();
// The default max cache size chosen for CUDA version > 10 is arbitrary.
// This number puts a limit on how big of a plan cache should we maintain by
// default. Users can always configure it via cufft_set_plan_cache_max_size.
constexpr size_t CUFFT_DEFAULT_CACHE_SIZE = 4096;
#endif
static_assert(CUFFT_MAX_PLAN_NUM >= 0 &&
                  CUFFT_MAX_PLAN_NUM <= std::numeric_limits<size_t>::max(),
              "CUFFT_MAX_PLAN_NUM not in size_t range");
static_assert(CUFFT_DEFAULT_CACHE_SIZE >= 0 &&
                  CUFFT_DEFAULT_CACHE_SIZE <= CUFFT_MAX_PLAN_NUM,
              "CUFFT_DEFAULT_CACHE_SIZE not in [0, CUFFT_MAX_PLAN_NUM] range");

// This cache assumes that the mapping from key to value never changes.
// This is **NOT** thread-safe. Please use a mutex when using it **AND** the
// value returned from try_emplace_value.
// The contract of using this cache is that try_emplace_value should only be
// used when the max_size is positive.
class PlanLRUCache {
 public:
  using kv_t = typename std::pair<PlanKey, CuFFTConfig>;
  using map_t =
      typename std::unordered_map<std::reference_wrapper<PlanKey>,
                                  typename std::list<kv_t>::iterator,
                                  KeyHash<PlanKey>, KeyEqual<PlanKey>>;
  using map_kkv_iter_t = typename map_t::iterator;

  PlanLRUCache() : PlanLRUCache(CUFFT_DEFAULT_CACHE_SIZE) {}

  explicit PlanLRUCache(int64_t max_size) { _set_max_size(max_size); }

  PlanLRUCache(PlanLRUCache&& other) noexcept
      : _usage_list(std::move(other._usage_list)),
        _cache_map(std::move(other._cache_map)),
        _max_size(other._max_size) {}

  PlanLRUCache& operator=(PlanLRUCache&& other) noexcept {
    _usage_list = std::move(other._usage_list);
    _cache_map = std::move(other._cache_map);
    _max_size = other._max_size;
    return *this;
  }

  // If key is in this cache, return the cached config. Otherwise, emplace the
  // config in this cache and return it.
  CuFFTConfig& lookup(PlanKey params) {
    PADDLE_ENFORCE_GT(_max_size, 0,
                      platform::errors::InvalidArgument(
                          "The max size of PlanLRUCache must be great than 0,"
                          "But received is [%d]",
                          _max_size));

    map_kkv_iter_t map_it = _cache_map.find(params);
    // Hit, put to list front
    if (map_it != _cache_map.end()) {
      _usage_list.splice(_usage_list.begin(), _usage_list, map_it->second);
      return map_it->second->second;
    }

    // Miss
    // remove if needed
    if (_usage_list.size() >= _max_size) {
      auto last = _usage_list.end();
      last--;
      _cache_map.erase(last->first);
      _usage_list.pop_back();
    }

    // construct new plan at list front, then insert into _cache_map
    _usage_list.emplace_front(std::piecewise_construct,
                              std::forward_as_tuple(params),
                              std::forward_as_tuple(params));
    auto kv_it = _usage_list.begin();
    _cache_map.emplace(std::piecewise_construct,
                       std::forward_as_tuple(kv_it->first),
                       std::forward_as_tuple(kv_it));
    return kv_it->second;
  }

  void clear() {
    _cache_map.clear();
    _usage_list.clear();
  }

  void resize(int64_t new_size) {
    _set_max_size(new_size);
    auto cur_size = _usage_list.size();
    if (cur_size > _max_size) {
      auto delete_it = _usage_list.end();
      for (size_t i = 0; i < cur_size - _max_size; i++) {
        delete_it--;
        _cache_map.erase(delete_it->first);
      }
      _usage_list.erase(delete_it, _usage_list.end());
    }
  }

  size_t size() const { return _cache_map.size(); }

  size_t max_size() const noexcept { return _max_size; }

  std::mutex mutex;

 private:
  // Only sets size and does value check. Does not resize the data structures.
  void _set_max_size(int64_t new_size) {
    // We check that 0 <= new_size <= CUFFT_MAX_PLAN_NUM here. Since
    // CUFFT_MAX_PLAN_NUM is of type size_t, we need to do non-negativity check
    // first.
    PADDLE_ENFORCE_GE(
        new_size, 0,
        platform::errors::InvalidArgument(
            "cuFFT plan cache size must be non-negative, But received is [%d]",
            new_size));
    PADDLE_ENFORCE_LE(new_size, CUFFT_MAX_PLAN_NUM,
                      platform::errors::InvalidArgument(
                          "cuFFT plan cache size can not be larger than [%d], "
                          "But received is [%d]",
                          CUFFT_MAX_PLAN_NUM, new_size));
    _max_size = static_cast<size_t>(new_size);
  }

  std::list<kv_t> _usage_list;
  map_t _cache_map;
  size_t _max_size;
};

// Execute a pre-planned transform
static void exec_cufft_plan(const CuFFTConfig& config, void* in_data,
                            void* out_data, bool forward) {
  auto& plan = config.plan();
#ifdef __HIPCC__
  auto value_type = config.data_type();
  if (value_type == framework::proto::VarType::FP32) {
    switch (config.transform_type()) {
      case FFTTransformType::C2C: {
        CUFFT_CHECK(hipfftExecC2C(plan, static_cast<hipfftComplex*>(in_data),
                                  static_cast<hipfftComplex*>(out_data),
                                  forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case FFTTransformType::R2C: {
        CUFFT_CHECK(hipfftExecR2C(plan, static_cast<hipfftReal*>(in_data),
                                  static_cast<hipfftComplex*>(out_data)));
        return;
      }
      case FFTTransformType::C2R: {
        CUFFT_CHECK(hipfftExecC2R(plan, static_cast<hipfftComplex*>(in_data),
                                  static_cast<hipfftReal*>(out_data)));
        return;
      }
    }
  } else if (value_type == framework::proto::VarType::FP64) {
    switch (config.transform_type()) {
      case FFTTransformType::C2C: {
        CUFFT_CHECK(hipfftExecZ2Z(plan,
                                  static_cast<hipfftDoubleComplex*>(in_data),
                                  static_cast<hipfftDoubleComplex*>(out_data),
                                  forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case FFTTransformType::R2C: {
        CUFFT_CHECK(hipfftExecD2Z(plan, static_cast<hipfftDoubleReal*>(in_data),
                                  static_cast<hipfftDoubleComplex*>(out_data)));
        return;
      }
      case FFTTransformType::C2R: {
        CUFFT_CHECK(hipfftExecZ2D(plan,
                                  static_cast<hipfftDoubleComplex*>(in_data),
                                  static_cast<hipfftDoubleReal*>(out_data)));
        return;
      }
    }
  }
  PADDLE_THROW(platform::errors::InvalidArgument(
      "hipFFT only support transforms of type float32 and float64"));
#else
  CUFFT_CHECK(hipfftXtExec(plan, in_data, out_data,
                          forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
#endif
}

static std::vector<std::unique_ptr<PlanLRUCache>> plan_caches;
static std::mutex plan_caches_mutex;

static inline PlanLRUCache& cufft_get_plan_cache(int64_t device_index) {
  std::lock_guard<std::mutex> guard(plan_caches_mutex);

  if (device_index >= plan_caches.size()) {
    plan_caches.resize(device_index + 1);
  }

  if (!plan_caches[device_index]) {
    plan_caches[device_index] = std::make_unique<PlanLRUCache>();
  }

  return *plan_caches[device_index];
}

// Execute a general unnormalized fft operation (can be c2c, onesided r2c or
// onesided c2r)
template <typename DeviceContext, typename Ti, typename To>
void exec_fft(const DeviceContext& ctx, const Tensor* X, Tensor* out,
              const std::vector<int64_t>& dim, bool forward) {
  const auto x_dims = framework::vectorize(X->dims());
  const auto out_dims = framework::vectorize(out->dims());
  const int64_t ndim = static_cast<int64_t>(X->dims().size());
  const int64_t signal_ndim = static_cast<int64_t>(dim.size());
  const int64_t batch_dims = ndim - signal_ndim;
  auto tensor_place = ctx.GetPlace();

  // Transpose batch dimensions first, then with transforming dims
  std::vector<int> dim_permute(ndim);
  std::vector<int> reverse_dim_permute(ndim);
  std::vector<int64_t> trans_dims(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int{0});
  std::vector<bool> is_transformed_dim(ndim);
  for (const auto& d : dim) {
    is_transformed_dim[d] = true;
  }
  auto batch_end =
      std::partition(dim_permute.begin(), dim_permute.end(),
                     [&](int64_t d) { return !is_transformed_dim[d]; });
  std::sort(dim_permute.begin(), batch_end);
  std::copy(dim.cbegin(), dim.cend(), batch_end);

  for (size_t i = 0; i < ndim; i++) {
    trans_dims[i] = x_dims[dim_permute[i]];  // shape of input transpose
    reverse_dim_permute[dim_permute[i]] =
        static_cast<int>(i);  // reverse of dim permute
  }
  framework::Tensor input;
  input.Resize(framework::make_ddim(trans_dims));
  input.mutable_data<Ti>(tensor_place);
  /*
  auto in_ret = TransposeSimple<Ti>::run(ctx, *X, dim_permute, input);
  if (!in_ret) {
    TransCompute<DeviceContext, Ti>(ndim, ctx, *X, input, dim_permute);
  }
  */
  TransCompute<DeviceContext, Ti>(ndim, ctx, *X, &input, dim_permute);

  // Reshape batch dimensions into a single dimension
  std::vector<int64_t> batched_sizes(signal_ndim + 1);
  auto batch_size =
      std::accumulate(trans_dims.begin(), trans_dims.begin() + batch_dims,
                      static_cast<int>(1), std::multiplies<int>());
  batched_sizes[0] = batch_size;
  std::copy(trans_dims.begin() + batch_dims, trans_dims.end(),
            batched_sizes.begin() + 1);
  input.Resize(framework::make_ddim(batched_sizes));

  // Check the shape of transforming dims with input and output
  std::vector<int64_t> signal_size(signal_ndim + 1);
  signal_size[0] = batch_size;
  for (int64_t i = 0; i < signal_ndim; ++i) {
    auto in_size = input.dims()[i + 1];
    auto out_size = out_dims[dim[i]];
    signal_size[i + 1] = std::max(in_size, out_size);
    PADDLE_ENFORCE_EQ(
        (in_size == signal_size[i + 1] ||
         in_size == (signal_size[i + 1] / 2) + 1),
        true,
        platform::errors::InvalidArgument(
            "The dimension[%d] of Input size: [%d] must be equal or half to "
            "The dimension[%d] of Output size: [%d]",
            dim[i], in_size, dim[i], out_size));
    PADDLE_ENFORCE_EQ(
        (out_size == signal_size[i + 1] ||
         out_size == (signal_size[i + 1] / 2) + 1),
        true,
        platform::errors::InvalidArgument(
            "The dimension[%d] of Output size: [%d] must be equal or half to "
            "The dimension[%d] of Input size: [%d]",
            dim[i], out_size, dim[i], in_size));
  }

  std::vector<int64_t> reshape_out_sizes(ndim);
  for (size_t i = 0; i < ndim; ++i) {
    reshape_out_sizes[i] = out_dims[dim_permute[i]];
  }
  std::vector<int64_t> batched_out_sizes(batched_sizes.begin(),
                                         batched_sizes.end());
  for (size_t i = 0; i < dim.size(); ++i) {
    batched_out_sizes[i + 1] = out_dims[dim[i]];
  }

  // output
  framework::Tensor output;
  output.Resize(framework::make_ddim(batched_out_sizes));
  output.mutable_data<To>(tensor_place);

  // Create the transform plan (either from cache or locally)
  const auto value_type = framework::IsComplexType(input.type())
                              ? framework::ToRealType(input.type())
                              : input.type();
  auto fft_type = GetFFTTransformType(input.type(), output.type());
  PlanKey Key(framework::vectorize(input.dims()),
              framework::vectorize(output.dims()), signal_size, fft_type,
              value_type);
  PlanLRUCache& plan_cache = cufft_get_plan_cache(static_cast<int64_t>(
      (reinterpret_cast<platform::CUDAPlace*>(&tensor_place))->GetDeviceId()));
  std::unique_lock<std::mutex> guard(plan_cache.mutex, std::defer_lock);
  CuFFTConfig* config = nullptr;

  if (plan_cache.max_size() > 0) {
    guard.lock();
    if (plan_cache.max_size() > 0) {  // check again after acquiring the lock
      config = &plan_cache.lookup(Key);
    }
  }

  if (config == nullptr) {
    CuFFTConfig uncached_plan(Key);
    config = &uncached_plan;
  }

  auto& plan = config->plan();

  // prepare cufft for execution
  CUFFT_CHECK(hipfftSetStream(plan, ctx.stream()));
  framework::Tensor workspace_tensor;
  workspace_tensor.mutable_data<To>(tensor_place, config->workspace_size());
  CUFFT_CHECK(hipfftSetWorkArea(plan, workspace_tensor.data<To>()));

  // execute transform plan
  if (fft_type == FFTTransformType::C2R && forward) {
    forward = false;
    framework::Tensor input_conj(input.type());
    input_conj.mutable_data<Ti>(input.dims(), ctx.GetPlace());
    platform::ForRange<DeviceContext> for_range(ctx, input.numel());
    math::ConjFunctor<Ti> functor(input.data<Ti>(), input.numel(),
                                  input_conj.data<Ti>());
    for_range(functor);
    exec_cufft_plan(*config, input_conj.data<void>(), output.data<void>(),
                    forward);
  } else if (fft_type == FFTTransformType::R2C && !forward) {
    forward = true;
    framework::Tensor out_conj(output.type());
    out_conj.mutable_data<To>(output.dims(), ctx.GetPlace());
    exec_cufft_plan(*config, input.data<void>(), out_conj.data<void>(),
                    forward);

    platform::ForRange<DeviceContext> for_range(ctx, output.numel());
    math::ConjFunctor<To> functor(out_conj.data<To>(), output.numel(),
                                  output.data<To>());
    for_range(functor);
  } else {
    exec_cufft_plan(*config, input.data<void>(), output.data<void>(), forward);
  }

  // Inverting output by reshape and transpose to original batch and dimension
  output.Resize(framework::make_ddim(reshape_out_sizes));
  out->Resize(framework::make_ddim(out_dims));
  TransCompute<DeviceContext, To>(ndim, ctx, output, out, reverse_dim_permute);
}

// Calculates the normalization constant
double fft_normalization_scale(FFTNormMode normalization,
                               const std::vector<int64_t>& sizes,
                               const std::vector<int64_t>& dims) {
  // auto norm = static_cast<fft_norm_mode>(normalization);
  if (normalization == FFTNormMode::none) {
    return static_cast<double>(1.0);
  }

  int64_t signal_numel = 1;
  for (auto dim : dims) {
    signal_numel *= sizes[dim];
  }
  const double scale_denom = (normalization == FFTNormMode::by_sqrt_n)
                                 ? std::sqrt(signal_numel)
                                 : static_cast<double>(signal_numel);
  return static_cast<double>(1.0 / scale_denom);
}

template <typename DeviceContext, typename T>
void exec_normalization(const DeviceContext& ctx, const Tensor* in, Tensor* out,
                        FFTNormMode normalization,
                        const std::vector<int64_t>& sizes,
                        const std::vector<int64_t>& axes) {
  double scale = fft_normalization_scale(normalization, sizes, axes);
  if (scale != 1.0) {
    auto eigen_out = framework::EigenVector<T>::Flatten(*out);
    auto eigen_in = framework::EigenVector<T>::Flatten(*in);
    auto dev = ctx.eigen_device();
    EigenScale<Eigen::GpuDevice, T>::Eval(*dev, eigen_out, eigen_in,
                                          static_cast<T>(scale),
                                          static_cast<T>(0), false);
  } else {
    framework::TensorCopy(*in, ctx.GetPlace(), out);
  }
}
}  // anonymous namespace

// Use the optimized path to perform single R2C or C2R if transformation dim is
// supported by cuFFT
bool use_optimized_cufft_path(const std::vector<int64_t>& axes) {
  // For performance reason, when axes starts with (0, 1), do not use the
  // optimized path.
  if (axes.size() > kMaxCUFFTNdim ||
      (axes.size() >= 2 && axes[0] == 0 && axes[1] == 1)) {
    return false;
  } else {
    return true;
  }
}

template <typename Ti, typename To>
struct FFTC2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    if (axes.empty()) {
      framework::TensorCopy(*X, ctx.GetPlace(), out);
      return;
    }

    framework::Tensor* p_out = out;
    std::vector<int64_t> out_dims = framework::vectorize(X->dims());
    std::vector<int64_t> working_axes(axes.begin(), axes.end());
    std::vector<int64_t> first_dims;
    size_t max_dims;
    framework::Tensor working_tensor;
    working_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
    framework::Tensor* p_working_tensor = &working_tensor;
    framework::TensorCopy(*X, ctx.GetPlace(), &working_tensor);

    while (true) {
      max_dims =
          std::min(static_cast<size_t>(kMaxCUFFTNdim), working_axes.size());
      first_dims.assign(working_axes.end() - max_dims, working_axes.end());

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, p_working_tensor,
                                                    p_out, first_dims, forward);
      working_axes.resize(working_axes.size() - max_dims);
      first_dims.clear();

      if (working_axes.empty()) {
        break;
      }

      std::swap(p_out, p_working_tensor);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, p_out, out, normalization, out_dims, axes);
  }
};

template <typename Ti, typename To>
struct FFTC2RFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    std::vector<int64_t> in_dims = framework::vectorize(X->dims());
    std::vector<int64_t> out_dims = framework::vectorize(out->dims());

    if (use_optimized_cufft_path(axes)) {
      framework::Tensor x_copy(X->type());
      x_copy.mutable_data<Ti>(X->dims(), ctx.GetPlace());
      framework::TensorCopy(*X, ctx.GetPlace(), &x_copy);
      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, &x_copy, out, axes,
                                                    forward);
    } else {
      framework::Tensor temp_tensor;
      temp_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
      const std::vector<int64_t> dims(axes.begin(), axes.end() - 1);

      FFTC2CFunctor<platform::CUDADeviceContext, Ti, Ti> c2c_functor;
      c2c_functor(ctx, X, &temp_tensor, dims, FFTNormMode::none, forward);

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, &temp_tensor, out,
                                                    {axes.back()}, forward);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, out, out, normalization, out_dims, axes);
  }
};

// n dimension real to complex FFT use cufft lib
template <typename Ti, typename To>
struct FFTR2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    // Step1: R2C transform on the last dimension
    framework::Tensor* r2c_out = out;
    const std::vector<int64_t> last_dim{axes.back()};
    std::vector<int64_t> out_dims = framework::vectorize(out->dims());
    exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, X, r2c_out, last_dim,
                                                  forward);

    // Step2: C2C transform on the remaining dimension
    framework::Tensor c2c_out;
    if (axes.size() > 1) {
      c2c_out.mutable_data<To>(out->dims(), ctx.GetPlace());
      std::vector<int64_t> remain_dim(axes.begin(), axes.end() - 1);
      FFTC2CFunctor<platform::CUDADeviceContext, To, To> fft_c2c_func;
      fft_c2c_func(ctx, r2c_out, &c2c_out, remain_dim, FFTNormMode::none,
                   forward);
    }

    const auto in_sizes = framework::vectorize(X->dims());
    framework::Tensor* norm_tensor = axes.size() > 1 ? &c2c_out : r2c_out;
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, norm_tensor, out, normalization, in_sizes, axes);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fft_c2c, ops::FFTC2CKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2CKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2c_grad,
    ops::FFTC2CGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2CGradKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2r, ops::FFTC2RKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2RKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2r_grad,
    ops::FFTC2RGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2RGradKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_r2c, ops::FFTR2CKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTR2CKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_r2c_grad,
    ops::FFTR2CGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTR2CGradKernel<paddle::platform::CUDADeviceContext, double>);
