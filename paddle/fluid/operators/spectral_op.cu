#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <functional>
#include <list>
#include <memory>
#include <mutex>
#include <numeric>
#include <sstream>
#include <stdexcept>
#include <string>
#include <unordered_map>
#include <vector>

#include "paddle/fluid/operators/conj_op.h"
#include "paddle/fluid/operators/spectral_op.h"
#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/dynload/hipfft/hipfft.h"

namespace paddle {
namespace operators {

namespace {

using ScalarType = framework::proto::VarType::Type;
const int64_t kMaxCUFFTNdim = 3;
const int64_t kMaxDataNdim = kMaxCUFFTNdim + 1;

static inline std::string get_cufft_error_info(hipfftResult error) {
  switch (error) {
    case HIPFFT_SUCCESS:
      return "HIPFFT_SUCCESS";
    case HIPFFT_INVALID_PLAN:
      return "HIPFFT_INVALID_PLAN";
    case HIPFFT_ALLOC_FAILED:
      return "HIPFFT_ALLOC_FAILED";
    case HIPFFT_INVALID_TYPE:
      return "HIPFFT_INVALID_TYPE";
    case HIPFFT_INVALID_VALUE:
      return "HIPFFT_INVALID_VALUE";
    case HIPFFT_INTERNAL_ERROR:
      return "HIPFFT_INTERNAL_ERROR";
    case HIPFFT_EXEC_FAILED:
      return "HIPFFT_EXEC_FAILED";
    case HIPFFT_SETUP_FAILED:
      return "HIPFFT_SETUP_FAILED";
    case HIPFFT_INVALID_SIZE:
      return "HIPFFT_INVALID_SIZE";
    case HIPFFT_UNALIGNED_DATA:
      return "HIPFFT_UNALIGNED_DATA";
    case HIPFFT_INCOMPLETE_PARAMETER_LIST:
      return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
    case HIPFFT_INVALID_DEVICE:
      return "HIPFFT_INVALID_DEVICE";
    case HIPFFT_PARSE_ERROR:
      return "HIPFFT_PARSE_ERROR";
    case HIPFFT_NO_WORKSPACE:
      return "HIPFFT_NO_WORKSPACE";
    case HIPFFT_NOT_IMPLEMENTED:
      return "HIPFFT_NOT_IMPLEMENTED";
#ifndef __HIPCC__
    case CUFFT_LICENSE_ERROR:
      return "CUFFT_LICENSE_ERROR";
#endif
    case HIPFFT_NOT_SUPPORTED:
      return "HIPFFT_NOT_SUPPORTED";
    default:
      std::ostringstream ss;
      ss << "unknown error " << error;
      return ss.str();
  }
}

static inline void CUFFT_CHECK(hipfftResult error) {
  if (error != HIPFFT_SUCCESS) {
    PADDLE_THROW(platform::errors::External(get_cufft_error_info(error)));
  }
}

// This struct is used to easily compute hashes of the
// parameters. It will be the **key** to the plan cache.
struct PlanKey {
  // between 1 and kMaxCUFFTNdim, i.e., 1 <= signal_ndim <= 3
  int64_t signal_ndim_;
  // These include additional batch dimension as well.
  int64_t sizes_[kMaxDataNdim];
  int64_t input_shape_[kMaxDataNdim];
  int64_t output_shape_[kMaxDataNdim];
  FFTTransformType fft_type_;
  ScalarType value_type_;

  PlanKey() = default;

  PlanKey(const std::vector<int64_t>& in_shape,
          const std::vector<int64_t>& out_shape,
          const std::vector<int64_t>& signal_size, FFTTransformType fft_type,
          ScalarType value_type) {
    // Padding bits must be zeroed for hashing
    memset(this, 0, sizeof(*this));
    signal_ndim_ = signal_size.size() - 1;
    fft_type_ = fft_type;
    value_type_ = value_type;

    std::copy(signal_size.cbegin(), signal_size.cend(), sizes_);
    std::copy(in_shape.cbegin(), in_shape.cend(), input_shape_);
    std::copy(out_shape.cbegin(), out_shape.cend(), output_shape_);
  }
};

// An RAII encapsulation of cuFFTHandle
class CuFFTHandle {
  ::hipfftHandle handle_;

 public:
  CuFFTHandle() { CUFFT_CHECK(platform::dynload::hipfftCreate(&handle_)); }

  ::hipfftHandle& get() { return handle_; }
  const ::hipfftHandle& get() const { return handle_; }

  ~CuFFTHandle() {
// Not using fftDestroy() for rocFFT to work around double freeing of handles
#ifndef __HIPCC__
    CUFFT_CHECK(platform::dynload::hipfftDestroy(handle_));
#endif
  }
};

#ifdef __HIPCC__
using plan_size_type = int;
#else
using plan_size_type = long long int;  // NOLINT
#endif

// This class contains all the information needed to execute a cuFFT plan:
//   1. the plan
//   2. the workspace size needed
class CuFFTConfig {
 public:
  // Only move semantics is enought for this class. Although we already use
  // unique_ptr for the plan, still remove copy constructor and assignment op so
  // we don't accidentally copy and take perf hit.
  CuFFTConfig(const CuFFTConfig&) = delete;
  CuFFTConfig& operator=(CuFFTConfig const&) = delete;

  explicit CuFFTConfig(const PlanKey& plan_key)
      : CuFFTConfig(
            std::vector<int64_t>(plan_key.sizes_,
                                 plan_key.sizes_ + plan_key.signal_ndim_ + 1),
            plan_key.signal_ndim_, plan_key.fft_type_, plan_key.value_type_) {}

  // sizes are full signal, including batch size and always two-sided
  CuFFTConfig(const std::vector<int64_t>& sizes, const int64_t signal_ndim,
              FFTTransformType fft_type, ScalarType dtype)
      : fft_type_(fft_type), value_type_(dtype) {
    // signal sizes (excluding batch dim)
    std::vector<plan_size_type> signal_sizes(sizes.begin() + 1, sizes.end());

    // input batch size
    const auto batch = static_cast<plan_size_type>(sizes[0]);
    // const int64_t signal_ndim = sizes.size() - 1;
    PADDLE_ENFORCE_EQ(signal_ndim, sizes.size() - 1,
                      platform::errors::InvalidArgument(
                          "The signal_ndim must be equal to sizes.size() - 1,"
                          "But signal_ndim is: [%d], sizes.size() - 1 is: [%d]",
                          signal_ndim, sizes.size() - 1));

#ifdef __HIPCC__
    hipfftType exec_type = [&] {
      if (dtype == framework::proto::VarType::FP32) {
        switch (fft_type) {
          case FFTTransformType::C2C:
            return HIPFFT_C2C;
          case FFTTransformType::R2C:
            return HIPFFT_R2C;
          case FFTTransformType::C2R:
            return HIPFFT_C2R;
        }
      } else if (dtype == framework::proto::VarType::FP64) {
        switch (fft_type) {
          case FFTTransformType::C2C:
            return HIPFFT_Z2Z;
          case FFTTransformType::R2C:
            return HIPFFT_D2Z;
          case FFTTransformType::C2R:
            return HIPFFT_Z2D;
        }
      }
      PADDLE_THROW(platform::errors::InvalidArgument(
          "hipFFT only support transforms of type float32 and float64"));
    }();
#else
    hipDataType itype, otype, exec_type;
    const auto complex_input = has_complex_input(fft_type);
    const auto complex_output = has_complex_output(fft_type);
    if (dtype == framework::proto::VarType::FP32) {
      itype = complex_input ? HIP_C_32F : HIP_R_32F;
      otype = complex_output ? HIP_C_32F : HIP_R_32F;
      exec_type = HIP_C_32F;
    } else if (dtype == framework::proto::VarType::FP64) {
      itype = complex_input ? HIP_C_64F : HIP_R_64F;
      otype = complex_output ? HIP_C_64F : HIP_R_64F;
      exec_type = HIP_C_64F;
    } else if (dtype == framework::proto::VarType::FP16) {
      itype = complex_input ? HIP_C_16F : HIP_R_16F;
      otype = complex_output ? HIP_C_16F : HIP_R_16F;
      exec_type = HIP_C_16F;
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "cuFFT only support transforms of type float16, float32 and "
          "float64"));
    }
#endif

    // disable auto allocation of workspace to use allocator from the framework
    CUFFT_CHECK(platform::dynload::hipfftSetAutoAllocation(
        plan(), /* autoAllocate */ 0));

    size_t ws_size_t;

// make plan
#ifdef __HIPCC__
    CUFFT_CHECK(hipfftMakePlanMany(
        plan(), signal_ndim, signal_sizes.data(),
        /* inembed */ nullptr, /* base_istride */ 1, /* idist */ 1,
        /* onembed */ nullptr, /* base_ostride */ 1, /* odist */ 1, exec_type,
        batch, &ws_size_t));
#else

    CUFFT_CHECK(platform::dynload::hipfftXtMakePlanMany(
        plan(), signal_ndim, signal_sizes.data(),
        /* inembed */ nullptr, /* base_istride */ 1, /* idist */ 1, itype,
        /* onembed */ nullptr, /* base_ostride */ 1, /* odist */ 1, otype,
        batch, &ws_size_t, exec_type));
#endif

    ws_size = ws_size_t;
  }

  const hipfftHandle& plan() const { return plan_ptr.get(); }

  FFTTransformType transform_type() const { return fft_type_; }
  ScalarType data_type() const { return value_type_; }
  size_t workspace_size() const { return ws_size; }

 private:
  CuFFTHandle plan_ptr;
  size_t ws_size;
  FFTTransformType fft_type_;
  ScalarType value_type_;
};

// Execute a pre-planned transform
static void exec_cufft_plan(const CuFFTConfig& config, void* in_data,
                            void* out_data, bool forward) {
  auto& plan = config.plan();
#ifdef __HIPCC__
  auto value_type = config.data_type();
  if (value_type == framework::proto::VarType::FP32) {
    switch (config.transform_type()) {
      case FFTTransformType::C2C: {
        CUFFT_CHECK(hipfftExecC2C(plan, static_cast<hipfftComplex*>(in_data),
                                  static_cast<hipfftComplex*>(out_data),
                                  forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case FFTTransformType::R2C: {
        CUFFT_CHECK(hipfftExecR2C(plan, static_cast<hipfftReal*>(in_data),
                                  static_cast<hipfftComplex*>(out_data)));
        return;
      }
      case FFTTransformType::C2R: {
        CUFFT_CHECK(hipfftExecC2R(plan, static_cast<hipfftComplex*>(in_data),
                                  static_cast<hipfftReal*>(out_data)));
        return;
      }
    }
  } else if (value_type == framework::proto::VarType::FP64) {
    switch (config.transform_type()) {
      case FFTTransformType::C2C: {
        CUFFT_CHECK(hipfftExecZ2Z(plan,
                                  static_cast<hipfftDoubleComplex*>(in_data),
                                  static_cast<hipfftDoubleComplex*>(out_data),
                                  forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case FFTTransformType::R2C: {
        CUFFT_CHECK(hipfftExecD2Z(plan, static_cast<hipfftDoubleReal*>(in_data),
                                  static_cast<hipfftDoubleComplex*>(out_data)));
        return;
      }
      case FFTTransformType::C2R: {
        CUFFT_CHECK(hipfftExecZ2D(plan,
                                  static_cast<hipfftDoubleComplex*>(in_data),
                                  static_cast<hipfftDoubleReal*>(out_data)));
        return;
      }
    }
  }
  PADDLE_THROW(platform::errors::InvalidArgument(
      "hipFFT only support transforms of type float32 and float64"));
#else
  CUFFT_CHECK(platform::dynload::hipfftXtExec(
      plan, in_data, out_data, forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
#endif
}

// Execute a general unnormalized fft operation (can be c2c, onesided r2c or
// onesided c2r)
template <typename DeviceContext, typename Ti, typename To>
void exec_fft(const DeviceContext& ctx, const Tensor* X, Tensor* out,
              const std::vector<int64_t>& dim, bool forward) {
  const auto x_dims = framework::vectorize(X->dims());
  const auto out_dims = framework::vectorize(out->dims());
  const int64_t ndim = static_cast<int64_t>(X->dims().size());
  const int64_t signal_ndim = static_cast<int64_t>(dim.size());
  const int64_t batch_dims = ndim - signal_ndim;
  auto tensor_place = ctx.GetPlace();

  // Transpose batch dimensions first, then with transforming dims
  std::vector<int> dim_permute(ndim);
  std::vector<int> reverse_dim_permute(ndim);
  std::vector<int64_t> trans_dims(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int{0});
  std::vector<bool> is_transformed_dim(ndim);
  for (const auto& d : dim) {
    is_transformed_dim[d] = true;
  }
  auto batch_end =
      std::partition(dim_permute.begin(), dim_permute.end(),
                     [&](int64_t d) { return !is_transformed_dim[d]; });
  std::sort(dim_permute.begin(), batch_end);
  std::copy(dim.cbegin(), dim.cend(), batch_end);

  for (size_t i = 0; i < ndim; i++) {
    trans_dims[i] = x_dims[dim_permute[i]];  // shape of input transpose
    reverse_dim_permute[dim_permute[i]] =
        static_cast<int>(i);  // reverse of dim permute
  }
  framework::Tensor input;
  input.Resize(framework::make_ddim(trans_dims));
  input.mutable_data<Ti>(tensor_place);
  /*
  auto in_ret = TransposeSimple<Ti>::run(ctx, *X, dim_permute, input);
  if (!in_ret) {
    TransCompute<DeviceContext, Ti>(ndim, ctx, *X, input, dim_permute);
  }
  */
  TransCompute<DeviceContext, Ti>(ndim, ctx, *X, &input, dim_permute);

  // Reshape batch dimensions into a single dimension
  std::vector<int64_t> batched_sizes(signal_ndim + 1);
  auto batch_size =
      std::accumulate(trans_dims.begin(), trans_dims.begin() + batch_dims,
                      static_cast<int>(1), std::multiplies<int>());
  batched_sizes[0] = batch_size;
  std::copy(trans_dims.begin() + batch_dims, trans_dims.end(),
            batched_sizes.begin() + 1);
  input.Resize(framework::make_ddim(batched_sizes));

  // Check the shape of transforming dims with input and output
  std::vector<int64_t> signal_size(signal_ndim + 1);
  signal_size[0] = batch_size;
  for (int64_t i = 0; i < signal_ndim; ++i) {
    auto in_size = input.dims()[i + 1];
    auto out_size = out_dims[dim[i]];
    signal_size[i + 1] = std::max(in_size, out_size);
    PADDLE_ENFORCE_EQ(
        (in_size == signal_size[i + 1] ||
         in_size == (signal_size[i + 1] / 2) + 1),
        true,
        platform::errors::InvalidArgument(
            "The dimension[%d] of Input size: [%d] must be equal or half to "
            "The dimension[%d] of Output size: [%d]",
            dim[i], in_size, dim[i], out_size));
    PADDLE_ENFORCE_EQ(
        (out_size == signal_size[i + 1] ||
         out_size == (signal_size[i + 1] / 2) + 1),
        true,
        platform::errors::InvalidArgument(
            "The dimension[%d] of Output size: [%d] must be equal or half to "
            "The dimension[%d] of Input size: [%d]",
            dim[i], out_size, dim[i], in_size));
  }

  std::vector<int64_t> reshape_out_sizes(ndim);
  for (size_t i = 0; i < ndim; ++i) {
    reshape_out_sizes[i] = out_dims[dim_permute[i]];
  }
  std::vector<int64_t> batched_out_sizes(batched_sizes.begin(),
                                         batched_sizes.end());
  for (size_t i = 0; i < dim.size(); ++i) {
    batched_out_sizes[i + 1] = out_dims[dim[i]];
  }

  // output
  framework::Tensor output;
  output.Resize(framework::make_ddim(batched_out_sizes));
  output.mutable_data<To>(tensor_place);

  // Create the transform plan (either from cache or locally)
  const auto value_type = framework::IsComplexType(input.type())
                              ? framework::ToRealType(input.type())
                              : input.type();
  auto fft_type = GetFFTTransformType(input.type(), output.type());
  PlanKey Key(framework::vectorize(input.dims()),
              framework::vectorize(output.dims()), signal_size, fft_type,
              value_type);
  CuFFTConfig uncached_plan(Key);
  CuFFTConfig* config = &uncached_plan;
  auto& plan = config->plan();

  // prepare cufft for execution
  CUFFT_CHECK(platform::dynload::hipfftSetStream(plan, ctx.stream()));
  framework::Tensor workspace_tensor;
  workspace_tensor.mutable_data<To>(tensor_place, config->workspace_size());
  CUFFT_CHECK(
      platform::dynload::hipfftSetWorkArea(plan, workspace_tensor.data<To>()));

  // execute transform plan
  if (fft_type == FFTTransformType::C2R && forward) {
    forward = false;
    framework::Tensor input_conj(input.type());
    input_conj.mutable_data<Ti>(input.dims(), ctx.GetPlace());
    platform::ForRange<DeviceContext> for_range(ctx, input.numel());
    math::ConjFunctor<Ti> functor(input.data<Ti>(), input.numel(),
                                  input_conj.data<Ti>());
    for_range(functor);
    exec_cufft_plan(*config, input_conj.data<void>(), output.data<void>(),
                    forward);
  } else if (fft_type == FFTTransformType::R2C && !forward) {
    forward = true;
    framework::Tensor out_conj(output.type());
    out_conj.mutable_data<To>(output.dims(), ctx.GetPlace());
    exec_cufft_plan(*config, input.data<void>(), out_conj.data<void>(),
                    forward);

    platform::ForRange<DeviceContext> for_range(ctx, output.numel());
    math::ConjFunctor<To> functor(out_conj.data<To>(), output.numel(),
                                  output.data<To>());
    for_range(functor);
  } else {
    exec_cufft_plan(*config, input.data<void>(), output.data<void>(), forward);
  }

  // Inverting output by reshape and transpose to original batch and dimension
  output.Resize(framework::make_ddim(reshape_out_sizes));
  out->Resize(framework::make_ddim(out_dims));
  TransCompute<DeviceContext, To>(ndim, ctx, output, out, reverse_dim_permute);
}

// Calculates the normalization constant
double fft_normalization_scale(FFTNormMode normalization,
                               const std::vector<int64_t>& sizes,
                               const std::vector<int64_t>& dims) {
  // auto norm = static_cast<fft_norm_mode>(normalization);
  if (normalization == FFTNormMode::none) {
    return static_cast<double>(1.0);
  }

  int64_t signal_numel = 1;
  for (auto dim : dims) {
    signal_numel *= sizes[dim];
  }
  const double scale_denom = (normalization == FFTNormMode::by_sqrt_n)
                                 ? std::sqrt(signal_numel)
                                 : static_cast<double>(signal_numel);
  return static_cast<double>(1.0 / scale_denom);
}

template <typename DeviceContext, typename T>
void exec_normalization(const DeviceContext& ctx, const Tensor* in, Tensor* out,
                        FFTNormMode normalization,
                        const std::vector<int64_t>& sizes,
                        const std::vector<int64_t>& axes) {
  double scale = fft_normalization_scale(normalization, sizes, axes);
  if (scale != 1.0) {
    auto eigen_out = framework::EigenVector<T>::Flatten(*out);
    auto eigen_in = framework::EigenVector<T>::Flatten(*in);
    auto dev = ctx.eigen_device();
    EigenScale<Eigen::GpuDevice, T>::Eval(*dev, eigen_out, eigen_in,
                                          static_cast<T>(scale),
                                          static_cast<T>(0), false);
  } else {
    framework::TensorCopy(*in, ctx.GetPlace(), out);
  }
}
}  // anonymous namespace

// Use the optimized path to perform single R2C or C2R if transformation dim is
// supported by cuFFT
bool use_optimized_cufft_path(const std::vector<int64_t>& axes) {
  // For performance reason, when axes starts with (0, 1), do not use the
  // optimized path.
  if (axes.size() > kMaxCUFFTNdim ||
      (axes.size() >= 2 && axes[0] == 0 && axes[1] == 1)) {
    return false;
  } else {
    return true;
  }
}

template <typename Ti, typename To>
struct FFTC2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    if (axes.empty()) {
      framework::TensorCopy(*X, ctx.GetPlace(), out);
      return;
    }

    framework::Tensor* p_out = out;
    std::vector<int64_t> out_dims = framework::vectorize(X->dims());
    std::vector<int64_t> working_axes(axes.begin(), axes.end());
    std::vector<int64_t> first_dims;
    size_t max_dims;
    framework::Tensor working_tensor;
    working_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
    framework::Tensor* p_working_tensor = &working_tensor;
    framework::TensorCopy(*X, ctx.GetPlace(), &working_tensor);

    while (true) {
      max_dims =
          std::min(static_cast<size_t>(kMaxCUFFTNdim), working_axes.size());
      first_dims.assign(working_axes.end() - max_dims, working_axes.end());

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, p_working_tensor,
                                                    p_out, first_dims, forward);
      working_axes.resize(working_axes.size() - max_dims);
      first_dims.clear();

      if (working_axes.empty()) {
        break;
      }

      std::swap(p_out, p_working_tensor);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, p_out, out, normalization, out_dims, axes);
  }
};

template <typename Ti, typename To>
struct FFTC2RFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    std::vector<int64_t> in_dims = framework::vectorize(X->dims());
    std::vector<int64_t> out_dims = framework::vectorize(out->dims());

    if (use_optimized_cufft_path(axes)) {
      framework::Tensor x_copy(X->type());
      x_copy.mutable_data<Ti>(X->dims(), ctx.GetPlace());
      framework::TensorCopy(*X, ctx.GetPlace(), &x_copy);
      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, &x_copy, out, axes,
                                                    forward);
    } else {
      framework::Tensor temp_tensor;
      temp_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
      const std::vector<int64_t> dims(axes.begin(), axes.end() - 1);

      FFTC2CFunctor<platform::CUDADeviceContext, Ti, Ti> c2c_functor;
      c2c_functor(ctx, X, &temp_tensor, dims, FFTNormMode::none, forward);

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, &temp_tensor, out,
                                                    {axes.back()}, forward);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, out, out, normalization, out_dims, axes);
  }
};

// n dimension real to complex FFT use cufft lib
template <typename Ti, typename To>
struct FFTR2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    // Step1: R2C transform on the last dimension
    framework::Tensor* r2c_out = out;
    const std::vector<int64_t> last_dim{axes.back()};
    std::vector<int64_t> out_dims = framework::vectorize(out->dims());
    exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, X, r2c_out, last_dim,
                                                  forward);

    // Step2: C2C transform on the remaining dimension
    framework::Tensor c2c_out;
    if (axes.size() > 1) {
      c2c_out.mutable_data<To>(out->dims(), ctx.GetPlace());
      std::vector<int64_t> remain_dim(axes.begin(), axes.end() - 1);
      FFTC2CFunctor<platform::CUDADeviceContext, To, To> fft_c2c_func;
      fft_c2c_func(ctx, r2c_out, &c2c_out, remain_dim, FFTNormMode::none,
                   forward);
    }

    const auto in_sizes = framework::vectorize(X->dims());
    framework::Tensor* norm_tensor = axes.size() > 1 ? &c2c_out : r2c_out;
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, norm_tensor, out, normalization, in_sizes, axes);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fft_c2c, ops::FFTC2CKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2CKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2c_grad,
    ops::FFTC2CGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2CGradKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2r, ops::FFTC2RKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2RKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2r_grad,
    ops::FFTC2RGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2RGradKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_r2c, ops::FFTR2CKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTR2CKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_r2c_grad,
    ops::FFTR2CGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTR2CGradKernel<paddle::platform::CUDADeviceContext, double>);
