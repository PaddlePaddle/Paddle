#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */
#include <functional>
#include <list>
#include <memory>
#include <mutex>
#include <numeric>
#include <sstream>
#include <stdexcept>
#include <string>
#include <unordered_map>
#include <vector>

#include "paddle/fluid/operators/conj_op.h"
#include "paddle/fluid/operators/spectral_helper.h"
#include "paddle/fluid/operators/spectral_op.h"
#include "paddle/fluid/operators/transpose_op.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"

namespace paddle {
namespace operators {

namespace {

// Calculates the normalization constant
double fft_normalization_scale(FFTNormMode normalization,
                               const std::vector<int64_t>& sizes,
                               const std::vector<int64_t>& dims) {
  // auto norm = static_cast<fft_norm_mode>(normalization);
  if (normalization == FFTNormMode::none) {
    return static_cast<double>(1.0);
  }

  int64_t signal_numel = 1;
  for (auto dim : dims) {
    signal_numel *= sizes[dim];
  }
  const double scale_denom = (normalization == FFTNormMode::by_sqrt_n)
                                 ? std::sqrt(signal_numel)
                                 : static_cast<double>(signal_numel);
  return static_cast<double>(1.0 / scale_denom);
}

template <typename DeviceContext, typename T>
void exec_normalization(const DeviceContext& ctx, const Tensor* in, Tensor* out,
                        FFTNormMode normalization,
                        const std::vector<int64_t>& sizes,
                        const std::vector<int64_t>& axes) {
  double scale = fft_normalization_scale(normalization, sizes, axes);
  if (scale != 1.0) {
    auto eigen_out = framework::EigenVector<T>::Flatten(*out);
    auto eigen_in = framework::EigenVector<T>::Flatten(*in);
    auto dev = ctx.eigen_device();
    EigenScale<Eigen::GpuDevice, T>::Eval(*dev, eigen_out, eigen_in,
                                          static_cast<T>(scale),
                                          static_cast<T>(0), false);
  } else {
    framework::TensorCopy(*in, ctx.GetPlace(), out);
  }
}

#if defined(PADDLE_WITH_CUDA)
FFTConfigKey create_fft_configkey(const framework::Tensor& input,
                                  const framework::Tensor& output,
                                  int signal_ndim) {
  // Create the transform plan (either from cache or locally)
  const auto value_type =
      framework::IsComplexType(framework::TransToProtoVarType(input.dtype()))
          ? framework::ToRealType(framework::TransToProtoVarType(input.dtype()))
          : framework::TransToProtoVarType(input.dtype());
  auto fft_type =
      GetFFTTransformType(framework::TransToProtoVarType(input.dtype()),
                          framework::TransToProtoVarType(output.dtype()));
  // signal sizes
  std::vector<int64_t> signal_size(signal_ndim + 1);

  signal_size[0] = input.dims()[0];
  for (int64_t i = 1; i <= signal_ndim; ++i) {
    auto in_size = input.dims()[i];
    auto out_size = output.dims()[i];
    signal_size[i] = std::max(in_size, out_size);
  }
  FFTConfigKey key(phi::vectorize(input.dims()), phi::vectorize(output.dims()),
                   signal_size, fft_type, value_type);
  return key;
}

// Execute a pre-planned transform
static void exec_cufft_plan_raw(const FFTConfig& config, void* in_data,
                                void* out_data, bool forward) {
  auto& plan = config.plan();

  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftXtExec(
      plan, in_data, out_data, forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
}

template <typename DeviceContext, typename Ti, typename To>
void exec_cufft_plan(const DeviceContext& ctx, const FFTConfig& config,
                     framework::Tensor* input, framework::Tensor* output,
                     bool forward) {
  // execute transform plan
  auto fft_type = config.transform_type();
  if (fft_type == FFTTransformType::C2R && forward) {
    forward = false;
    framework::Tensor input_conj(input->type());
    input_conj.mutable_data<Ti>(input->dims(), ctx.GetPlace());
    platform::ForRange<DeviceContext> for_range(ctx, input->numel());
    phi::funcs::ConjFunctor<Ti> functor(input->data<Ti>(), input->numel(),
                                        input_conj.data<Ti>());
    for_range(functor);
    exec_cufft_plan_raw(config, input_conj.data(), output->data(), forward);
  } else if (fft_type == FFTTransformType::R2C && !forward) {
    forward = true;
    framework::Tensor out_conj(output->type());
    out_conj.mutable_data<To>(output->dims(), ctx.GetPlace());
    exec_cufft_plan_raw(config, input->data(), out_conj.data(), forward);

    platform::ForRange<DeviceContext> for_range(ctx, output->numel());
    phi::funcs::ConjFunctor<To> functor(out_conj.data<To>(), output->numel(),
                                        output->data<To>());
    for_range(functor);
  } else {
    exec_cufft_plan_raw(config, input->data(), output->data(), forward);
  }
}

#elif defined(PADDLE_WITH_HIP)

FFTConfigKey create_fft_configkey(const framework::Tensor& input,
                                  const framework::Tensor& output,
                                  int signal_ndim) {
  // Create the transform plan (either from cache or locally)
  const auto value_type =
      framework::IsComplexType(framework::TransToProtoVarType(input.dtype()))
          ? framework::ToRealType(framework::TransToProtoVarType(input.dtype()))
          : framework::TransToProtoVarType(input.dtype());
  auto fft_type =
      GetFFTTransformType(framework::TransToProtoVarType(input.dtype()),
                          framework::TransToProtoVarType(output.type()));
  // signal sizes
  std::vector<int64_t> signal_size(signal_ndim + 1);

  signal_size[0] = input.dims()[0];
  for (int64_t i = 1; i <= signal_ndim; ++i) {
    auto in_size = input.dims()[i];
    auto out_size = output.dims()[i];
    signal_size[i] = std::max(in_size, out_size);
  }
  FFTConfigKey key(phi::vectorize(input.dims()), phi::vectorize(output.dims()),
                   signal_size, fft_type, value_type);
  return key;
}

// Execute a pre-planned transform
static void exec_hipfft_plan_raw(const FFTConfig& config, void* in_data,
                                 void* out_data, bool forward) {
  auto& plan = config.plan();

  auto value_type = config.data_type();
  if (value_type == framework::proto::VarType::FP32) {
    switch (config.transform_type()) {
      case FFTTransformType::C2C: {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftExecC2C(
            plan, static_cast<hipfftComplex*>(in_data),
            static_cast<hipfftComplex*>(out_data),
            forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case FFTTransformType::R2C: {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftExecR2C(
            plan, static_cast<hipfftReal*>(in_data),
            static_cast<hipfftComplex*>(out_data)));
        return;
      }
      case FFTTransformType::C2R: {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftExecC2R(
            plan, static_cast<hipfftComplex*>(in_data),
            static_cast<hipfftReal*>(out_data)));
        return;
      }
    }
  } else if (value_type == framework::proto::VarType::FP64) {
    switch (config.transform_type()) {
      case FFTTransformType::C2C: {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftExecZ2Z(
            plan, static_cast<hipfftDoubleComplex*>(in_data),
            static_cast<hipfftDoubleComplex*>(out_data),
            forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case FFTTransformType::R2C: {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftExecD2Z(
            plan, static_cast<hipfftDoubleReal*>(in_data),
            static_cast<hipfftDoubleComplex*>(out_data)));
        return;
      }
      case FFTTransformType::C2R: {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftExecZ2D(
            plan, static_cast<hipfftDoubleComplex*>(in_data),
            static_cast<hipfftDoubleReal*>(out_data)));
        return;
      }
    }
  }
  PADDLE_THROW(platform::errors::InvalidArgument(
      "hipFFT only support transforms of type float32 and float64"));
}

template <typename DeviceContext, typename Ti, typename To>
void exec_hipfft_plan(const DeviceContext& ctx, const FFTConfig& config,
                      framework::Tensor* input, framework::Tensor* output,
                      bool forward) {
  auto fft_type = config.transform_type();
  if (fft_type == FFTTransformType::C2R && forward) {
    forward = false;
    framework::Tensor input_conj(input->type());
    input_conj.mutable_data<Ti>(input->dims(), ctx.GetPlace());
    platform::ForRange<DeviceContext> for_range(ctx, input->numel());
    phi::funcs::ConjFunctor<Ti> functor(input->data<Ti>(), input->numel(),
                                        input_conj.data<Ti>());
    for_range(functor);
    exec_hipfft_plan_raw(config, input_conj.data(), output->data(), forward);
  } else if (fft_type == FFTTransformType::R2C && !forward) {
    forward = true;
    framework::Tensor out_conj(output->type());
    out_conj.mutable_data<To>(output->dims(), ctx.GetPlace());
    exec_hipfft_plan_raw(config, input->data(), out_conj.data(), forward);

    platform::ForRange<DeviceContext> for_range(ctx, output->numel());
    phi::funcs::ConjFunctor<To> functor(out_conj.data<To>(), output->numel(),
                                        output->data<To>());
    for_range(functor);
  } else {
    exec_hipfft_plan_raw(config, input->data(), output->data(), forward);
  }
}

#endif

// Execute a general unnormalized fft operation (can be c2c, onesided r2c or
// onesided c2r)
template <typename DeviceContext, typename Ti, typename To>
void exec_fft(const DeviceContext& ctx, const Tensor* X, Tensor* out,
              const std::vector<int64_t>& dim, bool forward) {
  const auto x_dims = phi::vectorize(X->dims());
  const int64_t ndim = static_cast<int64_t>(X->dims().size());
  auto tensor_place = ctx.GetPlace();

  // make a dim permutation
  std::vector<int> dim_permute(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int{0});
  std::vector<bool> is_transformed_dim(ndim);
  for (const auto& d : dim) {
    is_transformed_dim[d] = true;
  }
  auto batch_end =
      std::partition(dim_permute.begin(), dim_permute.end(),
                     [&](int64_t d) { return !is_transformed_dim[d]; });
  std::sort(dim_permute.begin(), batch_end);
  std::copy(dim.cbegin(), dim.cend(), batch_end);

  // transpose input according to dim permutation
  auto transposed_input_shape = X->dims().transpose(dim_permute);
  framework::Tensor transposed_input;
  transposed_input.Resize(transposed_input_shape);
  transposed_input.mutable_data<Ti>(tensor_place);
  TransCompute<DeviceContext, Ti>(ndim, ctx, *X, &transposed_input,
                                  dim_permute);

  // Reshape batch dimensions into a single dimension
  const int64_t signal_ndim = static_cast<int64_t>(dim.size());
  std::vector<int64_t> collapsed_input_shape(signal_ndim + 1);

  auto transposed_input_shape_ = phi::vectorize(transposed_input_shape);
  const int64_t batch_dims = ndim - signal_ndim;
  auto batch_size =
      std::accumulate(transposed_input_shape_.begin(),
                      transposed_input_shape_.begin() + batch_dims,
                      static_cast<int>(1), std::multiplies<int>());
  collapsed_input_shape[0] = batch_size;

  std::copy(transposed_input_shape_.begin() + batch_dims,
            transposed_input_shape_.end(), collapsed_input_shape.begin() + 1);

  framework::Tensor& collapsed_input = transposed_input;
  collapsed_input.Resize(phi::make_ddim(collapsed_input_shape));

  // make a collpased output
  const auto out_dims = phi::vectorize(out->dims());
  std::vector<int64_t> collapsed_output_shape(1 + signal_ndim);
  collapsed_output_shape[0] = batch_size;
  for (size_t i = 0; i < dim.size(); ++i) {
    collapsed_output_shape[i + 1] = out_dims[dim[i]];
  }
  framework::Tensor collapsed_output;
  collapsed_output.Resize(phi::make_ddim(collapsed_output_shape));
  collapsed_output.mutable_data<To>(tensor_place);

  FFTConfig* config = nullptr;

#if defined(PADDLE_WITH_CUDA)
  std::unique_ptr<FFTConfig> config_ = nullptr;
  // create plan
  FFTConfigKey key =
      create_fft_configkey(collapsed_input, collapsed_output, signal_ndim);
  bool using_cache = false;
#if !defined(CUFFT_VERSION) || (CUFFT_VERSION < 10200)
  using_cache = true;
#endif

  if (using_cache) {
    const int64_t device_id = static_cast<int64_t>(
        reinterpret_cast<const platform::CUDAPlace*>(&collapsed_input.place())
            ->GetDeviceId());
    FFTConfigCache& plan_cache = get_fft_plan_cache(device_id);
    std::unique_lock<std::mutex> guard(plan_cache.mutex, std::defer_lock);
    guard.lock();
    config = &(plan_cache.lookup(key));
  } else {
    config_ = std::make_unique<FFTConfig>(key);
    config = config_.get();
  }

  // prepare cufft for execution
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipfftSetStream(config->plan(), ctx.stream()));
  framework::Tensor workspace_tensor;
  workspace_tensor.mutable_data<To>(tensor_place, config->workspace_size());
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftSetWorkArea(
      config->plan(), workspace_tensor.data<To>()));
  // execute transform plan
  exec_cufft_plan<DeviceContext, Ti, To>(ctx, *config, &collapsed_input,
                                         &collapsed_output, forward);

#elif defined(PADDLE_WITH_HIP)
  // create plan
  FFTConfigKey key =
      create_fft_configkey(collapsed_input, collapsed_output, signal_ndim);
  const int64_t device_id = static_cast<int64_t>(
      reinterpret_cast<const platform::CUDAPlace*>(&collapsed_input.place())
          ->GetDeviceId());
  FFTConfigCache& plan_cache = get_fft_plan_cache(device_id);
  std::unique_lock<std::mutex> guard(plan_cache.mutex, std::defer_lock);
  guard.lock();
  config = &(plan_cache.lookup(key));

  // prepare cufft for execution
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipfftSetStream(config->plan(), ctx.stream()));
  framework::Tensor workspace_tensor;
  workspace_tensor.mutable_data<To>(tensor_place, config->workspace_size());
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipfftSetWorkArea(
      config->plan(), workspace_tensor.data<To>()));
  // execute transform plan
  exec_hipfft_plan<DeviceContext, Ti, To>(ctx, *config, &collapsed_input,
                                          &collapsed_output, forward);
#endif

  // Inverting output by reshape and transpose to original batch and dimension
  auto transposed_out_shape = out->dims().transpose(dim_permute);

  collapsed_output.Resize(transposed_out_shape);
  auto& transposed_output = collapsed_output;

  std::vector<int> reverse_dim_permute(ndim);
  for (size_t i = 0; i < ndim; i++) {
    reverse_dim_permute[dim_permute[i]] = i;
  }

  TransCompute<DeviceContext, To>(ndim, ctx, transposed_output, out,
                                  reverse_dim_permute);
}

}  // anonymous namespace

// Use the optimized path to perform single R2C or C2R if transformation dim is
// supported by cuFFT
bool use_optimized_fft_path(const std::vector<int64_t>& axes) {
  // For performance reason, when axes starts with (0, 1), do not use the
  // optimized path.
  if (axes.size() > kMaxFFTNdim ||
      (axes.size() >= 2 && axes[0] == 0 && axes[1] == 1)) {
    return false;
  } else {
    return true;
  }
}

template <typename Ti, typename To>
struct FFTC2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    if (axes.empty()) {
      framework::TensorCopy(*X, ctx.GetPlace(), out);
      return;
    }

    framework::Tensor* p_out = out;
    std::vector<int64_t> out_dims = phi::vectorize(X->dims());
    std::vector<int64_t> working_axes(axes.begin(), axes.end());
    std::vector<int64_t> first_dims;
    size_t max_dims;
    framework::Tensor working_tensor;
    working_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
    framework::Tensor* p_working_tensor = &working_tensor;
    framework::TensorCopy(*X, ctx.GetPlace(), &working_tensor);

    while (true) {
      max_dims =
          std::min(static_cast<size_t>(kMaxFFTNdim), working_axes.size());
      first_dims.assign(working_axes.end() - max_dims, working_axes.end());

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, p_working_tensor,
                                                    p_out, first_dims, forward);
      working_axes.resize(working_axes.size() - max_dims);
      first_dims.clear();

      if (working_axes.empty()) {
        break;
      }

      std::swap(p_out, p_working_tensor);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, p_out, out, normalization, out_dims, axes);
  }
};

template <typename Ti, typename To>
struct FFTC2RFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    std::vector<int64_t> in_dims = phi::vectorize(X->dims());
    std::vector<int64_t> out_dims = phi::vectorize(out->dims());

    if (use_optimized_fft_path(axes)) {
      framework::Tensor x_copy(X->type());
      x_copy.mutable_data<Ti>(X->dims(), ctx.GetPlace());
      framework::TensorCopy(*X, ctx.GetPlace(), &x_copy);
      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, &x_copy, out, axes,
                                                    forward);
    } else {
      framework::Tensor temp_tensor;
      temp_tensor.mutable_data<Ti>(X->dims(), ctx.GetPlace());
      const std::vector<int64_t> dims(axes.begin(), axes.end() - 1);

      FFTC2CFunctor<platform::CUDADeviceContext, Ti, Ti> c2c_functor;
      c2c_functor(ctx, X, &temp_tensor, dims, FFTNormMode::none, forward);

      exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, &temp_tensor, out,
                                                    {axes.back()}, forward);
    }
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, out, out, normalization, out_dims, axes);
  }
};

// n dimension real to complex FFT use cufft lib
template <typename Ti, typename To>
struct FFTR2CFunctor<platform::CUDADeviceContext, Ti, To> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor* X,
                  Tensor* out, const std::vector<int64_t>& axes,
                  FFTNormMode normalization, bool forward) {
    // Step1: R2C transform on the last dimension
    framework::Tensor* r2c_out = out;
    const std::vector<int64_t> last_dim{axes.back()};
    std::vector<int64_t> out_dims = phi::vectorize(out->dims());
    exec_fft<platform::CUDADeviceContext, Ti, To>(ctx, X, r2c_out, last_dim,
                                                  forward);

    // Step2: C2C transform on the remaining dimension
    framework::Tensor c2c_out;
    if (axes.size() > 1) {
      c2c_out.mutable_data<To>(out->dims(), ctx.GetPlace());
      std::vector<int64_t> remain_dim(axes.begin(), axes.end() - 1);
      FFTC2CFunctor<platform::CUDADeviceContext, To, To> fft_c2c_func;
      fft_c2c_func(ctx, r2c_out, &c2c_out, remain_dim, FFTNormMode::none,
                   forward);
    }

    const auto in_sizes = phi::vectorize(X->dims());
    framework::Tensor* norm_tensor = axes.size() > 1 ? &c2c_out : r2c_out;
    exec_normalization<platform::CUDADeviceContext, To>(
        ctx, norm_tensor, out, normalization, in_sizes, axes);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fft_c2c, ops::FFTC2CKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2CKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2c_grad,
    ops::FFTC2CGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2CGradKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2r, ops::FFTC2RKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2RKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_c2r_grad,
    ops::FFTC2RGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTC2RGradKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_r2c, ops::FFTR2CKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTR2CKernel<paddle::platform::CUDADeviceContext, double>);

REGISTER_OP_CUDA_KERNEL(
    fft_r2c_grad,
    ops::FFTR2CGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FFTR2CGradKernel<paddle::platform::CUDADeviceContext, double>);
