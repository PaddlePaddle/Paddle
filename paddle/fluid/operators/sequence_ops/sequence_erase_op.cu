#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "paddle/fluid/operators/sequence_ops/sequence_erase_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;
using LoDTensor = framework::LoDTensor;

template <typename T>
__global__ void LabelErasedIdx(const T* in_dat, const int64_t in_len,
                               const int* tokens, const size_t tokens_len,
                               size_t* num_erased) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_len) {
    for (size_t i = 0; i < tokens_len; ++i) {
      if (in_dat[index] == tokens[i]) {
        num_erased[index + 1] = 1;
        break;
      }
    }
  }
}

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod,
                          const size_t lod_len, size_t* out_lod0) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < lod_len) {
    out_lod0[index] = in_lod[index] - num_erased[in_lod[index]];
  }
}

template <typename T>
__global__ void SetOutput(const T* in_dat, const int64_t in_len,
                          const size_t* num_erased, T* out_dat) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_len) {
    if (num_erased[index] == num_erased[index + 1]) {
      out_dat[index - num_erased[index]] = in_dat[index];
    }
  }
}

template <typename T>
class SequenceEraseOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<LoDTensor>("X");
    auto* out = ctx.Output<LoDTensor>("Out");

    auto lod = in->lod();
    PADDLE_ENFORCE_EQ(lod[lod.size() - 1].back(), (size_t)in->numel(),
                      "The actual size mismatches with the LoD information.");
    auto tokens = ctx.Attr<std::vector<int>>("tokens");
    auto in_len = in->numel();
    auto in_dat = in->data<T>();
    // Copy tokens to GPU
    thrust::device_vector<int> dev_tokens(tokens.begin(), tokens.end());
    int* dev_tokens_ptr = thrust::raw_pointer_cast(dev_tokens.data());

    // Count number of elements to be erased
    thrust::device_vector<size_t> num_erased(in_len + 1, 0);
    size_t* num_erased_ptr = thrust::raw_pointer_cast(num_erased.data());
    auto stream = ctx.cuda_device_context().stream();
    LabelErasedIdx<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                     PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_dat, in_len, dev_tokens_ptr, tokens.size(), num_erased_ptr);
    thrust::inclusive_scan(num_erased.begin() + 1, num_erased.end(),
                           num_erased.begin() + 1);

    // Copy LoD to GPU
    auto last_lod = lod[lod.size() - 1];
    auto lod_len = last_lod.size();
    const size_t* dev_in_lod_ptr = last_lod.CUDAData(ctx.GetPlace());
    // Calc output LoD
    thrust::device_vector<size_t> dev_out_lod(lod_len);
    size_t* dev_out_lod_ptr = thrust::raw_pointer_cast(dev_out_lod.data());
    GetOutLod<<<(lod_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        num_erased_ptr, dev_in_lod_ptr, lod_len, dev_out_lod_ptr);
    // Set LoD for output
    std::vector<size_t> out_last_lod(dev_out_lod.begin(), dev_out_lod.end());
    framework::LoD out_lod;
    for (size_t i = 0; i < lod.size() - 1; ++i) {
      out_lod.push_back(lod[i]);
    }
    out_lod.push_back(out_last_lod);
    out->set_lod(out_lod);

    // Set output
    out->Resize({static_cast<int64_t>(out_last_lod.back()), 1});
    auto out_dat = out->mutable_data<T>(ctx.GetPlace());
    SetOutput<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_dat, in_len,
                                                      num_erased_ptr, out_dat);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(sequence_erase,
                        paddle::operators::SequenceEraseOpCUDAKernel<int32_t>,
                        paddle::operators::SequenceEraseOpCUDAKernel<int64_t>);
