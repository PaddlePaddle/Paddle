#include "hip/hip_runtime.h"
//   Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "paddle/fluid/operators/sequence_ops/sequence_enumerate_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;
using LoDTensor = framework::LoDTensor;

template <typename T>
__global__ void CalcOutPut(const T* in_data, const size_t* in_lod,
                           const size_t lod_len, const int64_t win_size,
                           const int64_t pad_value, T* out_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_lod[lod_len - 1]) {
    int end_idx = 0;
    // Get LoD interval of index
    for (int i = 1; i < lod_len; ++i) {
      if (index < in_lod[i]) {
        end_idx = in_lod[i];
        break;
      }
    }
    for (size_t i = 0; i < win_size; ++i) {
      int word_pos = index + i;
      out_data[index * win_size + i] =
          word_pos < end_idx ? in_data[word_pos] : pad_value;
    }
  }
}

template <typename T>
class SequenceEnumerateOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* out = context.Output<LoDTensor>("Out");
    int win_size = context.Attr<int>("win_size");
    int pad_value = context.Attr<int>("pad_value");

    auto in_dims = in->dims();
    auto in_lod = in->lod();

    PADDLE_ENFORCE_EQ(
        static_cast<uint64_t>(in_dims[0]), in_lod[0].back(),
        "The actual input data's size mismatched with LoD information.");

    /* Generate enumerate sequence set */
    auto stream = context.cuda_device_context().stream();
    auto lod0 = in_lod[0];
    auto in_len = in->numel();
    auto in_data = in->data<T>();
    out->Resize({in_dims[0], win_size});
    auto out_data = out->mutable_data<T>(context.GetPlace());
    // Copy LoD to GPU
    const size_t* dev_in_lod_ptr = lod0.CUDAData(context.GetPlace());
    // Calc output tensor
    CalcOutPut<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                 PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_data, dev_in_lod_ptr, lod0.size(), win_size, pad_value, out_data);
    out->set_lod(in->lod());
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    sequence_enumerate,
    paddle::operators::SequenceEnumerateOpCUDAKernel<int32_t>,
    paddle::operators::SequenceEnumerateOpCUDAKernel<int64_t>);
