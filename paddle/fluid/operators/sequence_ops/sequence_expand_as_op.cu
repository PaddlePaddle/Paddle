#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/operators/sequence_ops/sequence_expand_as_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using LoDTensor = framework::LoDTensor;

template <typename T>
static __global__ void sequence_expand_as_kernel(const T *in_data,
                                                 const size_t *expand_offset,
                                                 const size_t src_hight,
                                                 const size_t src_widht,
                                                 T *out_data) {
  for (int h_id = blockIdx.x; h_id < src_hight; h_id += gridDim.x) {
    int span = expand_offset[h_id + 1] - expand_offset[h_id];
    if (span == 0) continue;
    const T *src = in_data + h_id * src_widht;
    for (int w_id = threadIdx.x; w_id < src_widht; w_id += blockDim.x) {
      T ele = src[w_id];
      int offset = expand_offset[h_id] * src_widht;
      for (int k = 0; k < span; ++k) {
        out_data[offset + k * src_widht + w_id] = ele;
      }
    }
  }
}

template <typename T>
static __global__ void sequence_expand_as_grad_kernel(
    const T *dout_data, const size_t *expand_offset, const size_t dst_hight,
    const size_t dst_width, T *dx_data) {
  for (int h_id = blockIdx.x; h_id < dst_hight; h_id += gridDim.x) {
    T *dst = dx_data + h_id * dst_width;
    int span = expand_offset[h_id + 1] - expand_offset[h_id];

    for (int w_id = threadIdx.x; w_id < dst_width; w_id += blockDim.x) {
      T result = 0;
      for (int k = 0; k < span; ++k) {
        int offset = (expand_offset[h_id] + k) * dst_width;
        const T *src = dout_data + offset;
        result += src[w_id];
      }
      dst[w_id] = result;
    }
  }
}

template <typename T>
struct SequenceExpandFunctor<platform::CUDADeviceContext, T> {
  void operator()(
      const platform::CUDADeviceContext &context, const LoDTensor &x,
      const framework::Vector<size_t> &ref_lod, /*expand referenced lod*/
      LoDTensor *out) {
    int hight = x.dims()[0];
    int width = framework::product(x.dims()) / hight;

    const int kThreadsPerBlock = 1024;
    int thread_x = kThreadsPerBlock;
    if (width < kThreadsPerBlock) {  // block_cols is aligned by 32.
      thread_x = ((width + 31) >> 5) << 5;
    }

    int max_threads = context.GetMaxPhysicalThreadCount();
    int block_x = std::max(max_threads / thread_x, 1);

    dim3 block_size(thread_x);
    dim3 grid_size(block_x);
    sequence_expand_as_kernel<<<grid_size, block_size, 0, context.stream()>>>(
        x.data<T>(), ref_lod.CUDAData(context.GetPlace()), hight, width,
        out->mutable_data<T>(context.GetPlace()));
  }
};

template <typename T>
struct SequenceExpandAsGradFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext &context,
                  const LoDTensor &dout,
                  const framework::Vector<size_t> &ref_lod, /*expand based lod*/
                  LoDTensor *dx) {
    int hight = dx->dims()[0];
    int width = framework::product(dx->dims()) / hight;

    const int kThreadsPerBlock = 1024;
    int thread_x = kThreadsPerBlock;
    if (width < kThreadsPerBlock) {  // block_cols is aligned by 32.
      thread_x = ((width + 31) >> 5) << 5;
    }

    int max_threads = context.GetMaxPhysicalThreadCount();
    int block_x = std::max(max_threads / thread_x, 1);

    dim3 block_size(thread_x);
    dim3 grid_size(block_x);
    sequence_expand_as_grad_kernel<<<grid_size, block_size, 0,
                                     context.stream()>>>(
        dout.data<T>(), ref_lod.CUDAData(context.GetPlace()), hight, width,
        dx->mutable_data<T>(context.GetPlace()));
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    sequence_expand_as,
    ops::SequenceExpandAsKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SequenceExpandAsKernel<paddle::platform::CUDADeviceContext, double>,
    ops::SequenceExpandAsKernel<paddle::platform::CUDADeviceContext, int>,
    ops::SequenceExpandAsKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    sequence_expand_as_grad,
    ops::SequenceExpandAsGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SequenceExpandAsGradKernel<paddle::platform::CUDADeviceContext,
                                    double>,
    ops::SequenceExpandAsGradKernel<paddle::platform::CUDADeviceContext, int>,
    ops::SequenceExpandAsGradKernel<paddle::platform::CUDADeviceContext,
                                    int64_t>);
