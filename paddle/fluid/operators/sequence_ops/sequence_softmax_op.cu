#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <hipcub/hipcub.hpp>  // NOLINT
#include "paddle/fluid/operators/math.h"
#include "paddle/fluid/operators/sequence_ops/sequence_softmax_op.h"

namespace paddle {
namespace operators {

using LoDTensor = framework::LoDTensor;

template <typename T, int BlockDim>
using BlockReduce = hipcub::BlockReduce<T, BlockDim>;

template <typename T, int BlockDim>
using BlockReduceTempStorage = typename BlockReduce<T, BlockDim>::TempStorage;

template <typename T, int BlockDim>
__global__ void sequence_softmax_kernel(const T *in_data, const size_t *ref_lod,
                                        const size_t src_hight, T *out_data) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;
  __shared__ T shared_max_data;
  __shared__ T shared_sum_data;

  for (int i = blockIdx.x; i < src_hight; i += gridDim.x) {
    size_t start = ref_lod[i];
    size_t span = ref_lod[i + 1] - start;

    // Find the max ele
    T max_ele = -FLT_MAX;
    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      T ele = in_data[start + tid];
      max_ele = max_ele > ele ? max_ele : ele;
    }
    max_ele =
        BlockReduce<T, BlockDim>(temp_storage).Reduce(max_ele, hipcub::Max());
    if (threadIdx.x == 0) {
      shared_max_data = max_ele;
    }
    __syncthreads();

    // sum
    T sum_data = 0;
    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      T ele = in_data[start + tid];
      sum_data += real_exp(ele - shared_max_data);
    }
    sum_data =
        BlockReduce<T, BlockDim>(temp_storage).Reduce(sum_data, hipcub::Sum());
    if (threadIdx.x == 0) {
      shared_sum_data = sum_data;
    }
    __syncthreads();

    // get final resit
    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      T ele = in_data[start + tid];
      ele = real_exp(ele - shared_max_data) / shared_sum_data;
      out_data[start + tid] = ele;
    }
  }
}

template <typename T, int BlockDim>
__global__ void sequence_softmax_grad_kernel(const T *softmax_grad_data,
                                             const T *softmax_data,
                                             const size_t *ref_lod,
                                             const size_t src_hight,
                                             T *dx_data) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;
  __shared__ T shared_data;

  for (int i = blockIdx.x; i < src_hight; i += gridDim.x) {
    size_t start = ref_lod[i];
    size_t span = ref_lod[i + 1] - start;

    T result = 0;
    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      size_t idx = start + tid;
      T s_g_d = softmax_grad_data[idx];
      T s_d = softmax_data[idx];
      result += s_g_d * s_d;
    }
    result = BlockReduce<T, BlockDim>(temp_storage).Reduce(result, hipcub::Sum());
    if (threadIdx.x == 0) {
      shared_data = result;
    }
    __syncthreads();

    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      size_t idx = start + tid;
      T s_g_d = softmax_grad_data[idx];
      T s_d = softmax_data[idx];
      dx_data[idx] = (s_g_d - shared_data) * s_d;
    }
  }
}

template <typename T>
struct SequenceSoftmaxFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext &context,
                  const LoDTensor &x,
                  const framework::Vector<size_t> &ref_lod, /*referenced lod*/
                  LoDTensor *out) {
    int hight = ref_lod.size() - 1;

    const int kThreadsPerBlock = 32;
    int thread_x = kThreadsPerBlock;
    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    dim3 block_size(thread_x);
    dim3 grid_size(max_blocks);
    sequence_softmax_kernel<
        T, kThreadsPerBlock><<<grid_size, block_size, 0, context.stream()>>>(
        x.data<T>(), ref_lod.CUDAData(context.GetPlace()), hight,
        out->mutable_data<T>(context.GetPlace()));
  }
};

template <typename T>
struct SequenceSoftmaxGradFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext &context,
                  const LoDTensor &dout, const LoDTensor &out,
                  const framework::Vector<size_t> &ref_lod, /*referenced lod*/
                  LoDTensor *dx) {
    size_t hight = ref_lod.size() - 1;

    const int kThreadsPerBlock = 32;
    int thread_x = kThreadsPerBlock;
    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    dim3 block_size(thread_x);
    dim3 grid_size(max_blocks);

    sequence_softmax_grad_kernel<
        T, kThreadsPerBlock><<<grid_size, block_size, 0, context.stream()>>>(
        dout.data<T>(), out.data<T>(), ref_lod.CUDAData(context.GetPlace()),
        hight, dx->mutable_data<T>(context.GetPlace()));
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    sequence_softmax,
    ops::SequenceSoftmaxKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SequenceSoftmaxKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    sequence_softmax_grad,
    ops::SequenceSoftmaxGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::SequenceSoftmaxGradKernel<paddle::platform::CUDADeviceContext,
                                   double>);
