#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_functor.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/viterbi_decode_op.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

namespace paddle {
namespace operators {

#define CUDA_ARGMAX(kBlockDim)                                           \
  do {                                                                   \
    ArgmaxCUDAKernel<T, IndType,                                         \
                     kBlockDim><<<grid_size, kBlockDim, 0, cu_stream>>>( \
        height, width, post, in_data, out_idx_data, out_data);           \
  } while (0)

template <template <typename T> typename BinaryFunctor, typename T>
struct BinaryOperation<platform::CUDADeviceContext, BinaryFunctor, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx, const Tensor& lhs,
                  const Tensor& rhs, Tensor* output) {
    std::vector<const Tensor*> ins{&lhs, &rhs};
    std::vector<Tensor*> outs{output};
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        dev_ctx, ins, &outs, -1, BinaryFunctor<T>());
  }
};

template <template <typename T> typename CompareFunctor, typename T>
struct GetMask<platform::CUDADeviceContext, CompareFunctor, T> {
  void operator()(const framework::ExecutionContext& ctx, const Tensor& lhs,
                  const Tensor& rhs, Tensor* mask) {
    std::vector<const Tensor*> ins = {&lhs, &rhs};
    std::vector<Tensor*> outs = {mask};
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, int64_t, T>(
        dev_ctx, ins, &outs, -1, CompareFunctor<int64_t>());
  }
};

template <typename T, typename IndType, size_t BlockDim>
__global__ void ArgmaxCUDAKernel(const int64_t height,     // n * h
                                 const int64_t width,      // c
                                 const int64_t post_size,  // h
                                 const T* in, IndType* out_idx, T* out) {
  typedef hipcub::BlockReduce<hipcub::KeyValuePair<int, T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  hipcub::ArgMax reducer;
  T init = (std::numeric_limits<T>::lowest)();  // for windows compile
  for (int idx = blockIdx.x; idx < height; idx += gridDim.x) {
    hipcub::KeyValuePair<int, T> kv_pair = {-1, init};
    int h = idx / post_size;
    int w = idx % post_size;
    for (int k = threadIdx.x; k < width; k += blockDim.x) {
      kv_pair =
          reducer({k, in[h * width * post_size + k * post_size + w]}, kv_pair);
    }
    kv_pair = BlockReduce(temp_storage).Reduce(kv_pair, reducer);
    if (threadIdx.x == 0) {
      // return max, argmax
      if (out_idx != nullptr) out_idx[idx] = static_cast<IndType>(kv_pair.key);
      if (out != nullptr) out[idx] = kv_pair.value;
    }
    __syncthreads();
  }
}

__global__ void ARangeKernel(int64_t* data, int end, int64_t scale) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int start = idx; idx < end; idx += gridDim.x) {
    data[idx] = idx * scale;
  }
}

template <>
struct ARange<platform::CUDADeviceContext> {
  void operator()(const platform::CUDADeviceContext& dev_ctx, int64_t* data,
                  int end, int64_t scale) {
    ARangeKernel<<<1, 128, 0, dev_ctx.stream()>>>(data, end, scale);
  }
};

template <typename T, typename IndType>
struct Argmax<platform::CUDADeviceContext, T, IndType> {
  void operator()(const framework::ExecutionContext& ctx, const Tensor& input,
                  Tensor* out_idx, Tensor* out, int axis) {
    framework::DDim input_dims = input.dims();
    int64_t numel = input.numel();
    int64_t groups = numel / input_dims[axis];
    int64_t pre = 1;
    int64_t post = 1;
    int64_t n = input_dims[axis];
    for (int i = 0; i < axis; i++) {
      pre *= input_dims[i];
    }
    for (int i = axis + 1; i < input_dims.size(); i++) {
      post *= input_dims[i];
    }
    const auto& dev_ctx = ctx.cuda_device_context();
    auto cu_stream = dev_ctx.stream();
    int64_t max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize().x;
    int64_t height = pre * post;
    int64_t width = n;
    int64_t grid_size = height < max_grid_dimx ? height : max_grid_dimx;

    const T* in_data = input.data<T>();
    IndType* out_idx_data = out_idx->data<IndType>();
    T* out_data = out->data<T>();
    CUDA_ARGMAX(128);
  }
};

template <typename T>
struct GetMaxValue<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const Tensor& input, T* max_value) {
    Tensor out_data;
    out_data.Resize(framework::make_ddim({1}));
    out_data.mutable_data<T>(platform::CUDAPlace());
    ArgmaxCUDAKernel<T, T, 32><<<1, 32, 0, dev_ctx.stream()>>>(
        1, input.numel(), 1, input.data<int64_t>(), nullptr,
        out_data.data<int64_t>());
    Tensor max_value_tensor;
    framework::TensorCopy(out_data, platform::CPUPlace(), &max_value_tensor);
    *max_value = max_value_tensor.data<T>()[0];
  }
};

template <typename T, typename IndexT>
struct Gather<platform::CUDADeviceContext, T, IndexT> {
  void operator()(const platform::CUDADeviceContext& ctx, const Tensor& src,
                  const Tensor& index, Tensor* output) {
    GPUGather<T, IndexT>(ctx, src, index, output);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace platform = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    viterbi_decode,
    ops::ViterbiDecodeKernel<platform::CUDADeviceContext, float>,
    ops::ViterbiDecodeKernel<platform::CUDADeviceContext, double>);
