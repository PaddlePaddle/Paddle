#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/elementwise/elementwise_functor.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_broadcast.cu.h"
#include "paddle/fluid/operators/viterbi_decode_op.h"
#include "paddle/phi/kernels/funcs/gather.cu.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

namespace paddle {
namespace operators {

#define FIXED_BLOCK_DIM_CASE_BASE(log2_block_dim, ...)  \
  case (1 << (log2_block_dim)): {                       \
    constexpr auto kBlockDim = (1 << (log2_block_dim)); \
    __VA_ARGS__;                                        \
  } break

#define FIXED_BLOCK_DIM_CASE(...)               \
  FIXED_BLOCK_DIM_CASE_BASE(10, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_CASE_BASE(9, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_CASE_BASE(8, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_CASE_BASE(7, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_CASE_BASE(6, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_CASE_BASE(5, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_CASE_BASE(4, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_CASE_BASE(3, ##__VA_ARGS__);

int64_t ComputeBlockSize(int64_t col) {
  if (col > 512)
    return 1024;
  else if (col > 256)
    return 512;
  else if (col > 128)
    return 256;
  else if (col > 64)
    return 128;
  else if (col > 32)
    return 64;
  else if (col > 16)
    return 32;
  else if (col > 8)
    return 16;
  else
    return 8;
}

template <template <typename T> typename BinaryFunctor, typename T>
struct BinaryOperation<platform::CUDADeviceContext, BinaryFunctor, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor& lhs, const framework::Tensor& rhs,
                  framework::Tensor* output) {
    std::vector<const framework::Tensor*> ins{&lhs, &rhs};
    std::vector<framework::Tensor*> outs{output};
    paddle::operators::LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T,
                                                   T>(dev_ctx, ins, &outs, -1,
                                                      BinaryFunctor<T>());
  }
};

template <template <typename InT, typename OutT> typename CompareFunctor,
          typename T>
struct GetMask<platform::CUDADeviceContext, CompareFunctor, T> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor& lhs, const framework::Tensor& rhs,
                  framework::Tensor* mask) {
    std::vector<const framework::Tensor*> ins = {&lhs, &rhs};
    std::vector<framework::Tensor*> outs = {mask};
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    paddle::operators::LaunchSameDimsElementwiseCudaKernel<T>(
        dev_ctx, ins, &outs, CompareFunctor<int64_t, T>());
  }
};

template <typename T, typename IndType, size_t BlockDim>
__global__ void ArgmaxCUDAKernel(const int64_t height,     // n * h
                                 const int64_t width,      // c
                                 const int64_t post_size,  // h
                                 const T* in, IndType* out_idx, T* out) {
  typedef hipcub::BlockReduce<hipcub::KeyValuePair<int, T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  hipcub::ArgMax reducer;
  T init = (std::numeric_limits<T>::lowest)();  // for windows compile
  for (int idx = blockIdx.x; idx < height; idx += gridDim.x) {
    hipcub::KeyValuePair<int, T> kv_pair = {-1, init};
    int h = idx / post_size;
    int w = idx % post_size;
    for (int k = threadIdx.x; k < width; k += blockDim.x) {
      kv_pair =
          reducer({k, in[h * width * post_size + k * post_size + w]}, kv_pair);
    }
    kv_pair = BlockReduce(temp_storage).Reduce(kv_pair, reducer);
    if (threadIdx.x == 0) {
      // return max, argmax
      if (out_idx != nullptr) out_idx[idx] = static_cast<IndType>(kv_pair.key);
      if (out != nullptr) out[idx] = kv_pair.value;
    }
    __syncthreads();
  }
}

__global__ void ARangeKernel(int64_t* data, int num, int64_t scale) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int start = idx; idx < num; idx += gridDim.x) {
    data[idx] = idx * scale;
  }
}

template <>
struct ARange<platform::CUDADeviceContext> {
  void operator()(const platform::CUDADeviceContext& dev_ctx, int64_t* data,
                  int num, int64_t scale) {
    int64_t kBlockDim = ComputeBlockSize(num);
    // kBlockDim > num at most of time, so we can set grid = 1
    ARangeKernel<<<1, kBlockDim, 0, dev_ctx.stream()>>>(data, num, scale);
  }
};

template <typename T, typename IndType>
struct Argmax<platform::CUDADeviceContext, T, IndType> {
  void operator()(const framework::ExecutionContext& ctx,
                  const framework::Tensor& input, framework::Tensor* out_idx,
                  framework::Tensor* out, int axis) {
    framework::DDim input_dims = input.dims();
    int64_t numel = input.numel();
    int64_t groups = numel / input_dims[axis];
    int64_t pre = 1;
    int64_t post = 1;
    int64_t n = input_dims[axis];
    for (int i = 0; i < axis; i++) {
      pre *= input_dims[i];
    }
    for (int i = axis + 1; i < input_dims.size(); i++) {
      post *= input_dims[i];
    }
    const auto& dev_ctx = ctx.cuda_device_context();
    auto cu_stream = dev_ctx.stream();
    int64_t max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
    int64_t height = pre * post;
    int64_t width = n;
    int64_t grid_size = height < max_grid_dimx ? height : max_grid_dimx;
    const T* in_data = input.data<T>();
    IndType* out_idx_data = out_idx->data<IndType>();
    T* out_data = out->data<T>();
    switch (ComputeBlockSize(width)) {
      FIXED_BLOCK_DIM_CASE(
          ArgmaxCUDAKernel<T, IndType,
                           kBlockDim><<<grid_size, kBlockDim, 0, cu_stream>>>(
              height, width, post, in_data, out_idx_data, out_data));
    }
  }
};

template <typename T>
struct GetMaxValue<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor& input, T* max_value) {
    framework::Tensor out_data;
    out_data.Resize(phi::make_ddim({1}));
    out_data.mutable_data<T>(platform::CUDAPlace());
    switch (ComputeBlockSize(input.numel())) {
      FIXED_BLOCK_DIM_CASE(
          ArgmaxCUDAKernel<T, T,
                           kBlockDim><<<1, kBlockDim, 0, dev_ctx.stream()>>>(
              1, input.numel(), 1, input.data<int64_t>(), nullptr,
              out_data.data<int64_t>()));
    }
    framework::Tensor max_value_tensor;
    framework::TensorCopy(out_data, platform::CPUPlace(), &max_value_tensor);
    *max_value = max_value_tensor.data<T>()[0];
  }
};

template <typename T, typename IndexT>
struct Gather<platform::CUDADeviceContext, T, IndexT> {
  void operator()(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor& src, const framework::Tensor& index,
                  framework::Tensor* output) {
    phi::funcs::GPUGather<T, IndexT>(ctx, src, index, output);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace platform = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    viterbi_decode,
    ops::ViterbiDecodeKernel<platform::CUDADeviceContext, float>,
    ops::ViterbiDecodeKernel<platform::CUDADeviceContext, double>);
