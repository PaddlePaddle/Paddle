#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/flip_op.h"

#include <vector>
#include "paddle/fluid/memory/malloc.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using CUDADeviceContext = paddle::platform::CUDADeviceContext;

template <typename T>
__global__ void kernel_pointwise_flip_apply(const int N, const T* in_data,
                                            T* out_data, int dim0, int stride0,
                                            int dim1, int flip_dim) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N;
       idx += gridDim.x * blockDim.x) {
    int dst_offset = 0;
    if (flip_dim == 0) {
      // flip 1st dim
      dst_offset = (dim0 - 1 - idx / stride0) * stride0 + idx % stride0;
    } else {
      // flip last dim
      dst_offset = idx / stride0 * stride0 + (dim1 - 1 - idx % stride0);
    }
    out_data[dst_offset] = in_data[idx];
  }
}

template <typename T>
__global__ void flip_cuda_kernel(const int N, const T* in_data, T* out_data,
                                 int64_t* x_shape, int64_t* x_stride,
                                 int* flip_dims, int flip_dims_size,
                                 int total_dims) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int cur_indices = idx, rem = 0, dst_offset = 0;
  for (int i = 0; i < total_dims; ++i) {
    int64_t temp = cur_indices;
    cur_indices = cur_indices / x_stride[i];
    rem = temp - cur_indices * x_stride[i];
    // flip the indices if it is in flip_dims
    for (int j = 0; j < flip_dims_size; ++j) {
      if (i == flip_dims[j]) {
        cur_indices = x_shape[i] - 1 - cur_indices;
      }
    }
    dst_offset += cur_indices * x_stride[i];
    cur_indices = rem;
  }
  out_data[idx] = in_data[dst_offset];
}

template <typename T>
class FlipKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const auto gplace = BOOST_GET_CONST(platform::CUDAPlace, ctx.GetPlace());
    auto cplace = platform::CPUPlace();
    auto& dev_ctx = ctx.template device_context<CUDADeviceContext>();

    const Tensor* x = ctx.Input<Tensor>("X");
    Tensor* out = ctx.Output<Tensor>("Out");
    auto* in_data = x->data<T>();
    auto* out_data = out->mutable_data<T>(ctx.GetPlace());
    auto flip_dims = ctx.template Attr<std::vector<int>>("dims");

    const int flip_dims_size = static_cast<int>(flip_dims.size());
    auto x_dims = x->dims();
    const int total_dims = x_dims.size();
    const int N = x->numel();

    int block_size = 512;
    dim3 dim_block(block_size);
    dim3 dim_grid((N + block_size - 1) / block_size);

    for (size_t i = 0; i < flip_dims.size(); ++i) {
      if (flip_dims[i] < 0) {
        flip_dims[i] += total_dims;
      }
    }

    auto x_stride = framework::stride(x_dims);
    std::vector<int64_t> x_dims_v = framework::vectorize(x_dims);
    std::vector<int64_t> x_stride_v = framework::vectorize(x_stride);

    // wrap high-dims to 2-dims
    if (flip_dims_size == 1 &&
        (flip_dims[0] == 0 || flip_dims[0] == total_dims - 1)) {
      int dim0 = 1, dim1 = 1;
      int stride0 = 1;
      if (flip_dims[0] == 0) {
        dim0 = x_dims_v[0];
        stride0 = x_stride_v[0];
        for (size_t i = 1; i < total_dims; ++i) {
          dim1 *= x_dims_v[i];
        }
      } else {
        dim1 = x_dims_v[total_dims - 1];
        for (size_t i = 0; i < total_dims - 1; ++i) {
          dim0 *= x_dims_v[i];
        }
        stride0 *= x_dims_v[total_dims - 1];
      }
      kernel_pointwise_flip_apply<
          T><<<dim_grid, dim_block, 0, ctx.cuda_device_context().stream()>>>(
          N, in_data, out_data, dim0, stride0, dim1, flip_dims[0]);
    }

    int bytes = total_dims * sizeof(int64_t);
    auto x_strides_array_tmp = memory::Alloc(dev_ctx, bytes);
    int64_t* x_strides_array_gpu =
        reinterpret_cast<int64_t*>(x_strides_array_tmp->ptr());
    memory::Copy(gplace, x_strides_array_gpu, cplace, x_stride_v.data(), bytes,
                 dev_ctx.stream());

    auto x_shape_array_tmp = memory::Alloc(dev_ctx, bytes);
    int64_t* x_shape_array_gpu =
        reinterpret_cast<int64_t*>(x_shape_array_tmp->ptr());
    memory::Copy(gplace, x_shape_array_gpu, cplace, x_dims_v.data(), bytes,
                 dev_ctx.stream());

    bytes = flip_dims_size * sizeof(int);
    auto flip_dims_array_tmp = memory::Alloc(dev_ctx, bytes);
    int* flip_dims_array_gpu =
        reinterpret_cast<int*>(flip_dims_array_tmp->ptr());
    memory::Copy(gplace, flip_dims_array_gpu, cplace, flip_dims.data(), bytes,
                 dev_ctx.stream());

    flip_cuda_kernel<
        T><<<dim_grid, dim_block, 0, ctx.cuda_device_context().stream()>>>(
        N, in_data, out_data, x_shape_array_gpu, x_strides_array_gpu,
        flip_dims_array_gpu, flip_dims_size, total_dims);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    flip, ops::FlipKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FlipKernel<paddle::platform::CUDADeviceContext, double>,
    ops::FlipKernel<paddle::platform::CUDADeviceContext, plat::float16>,
    ops::FlipKernel<paddle::platform::CUDADeviceContext, int>,
    ops::FlipKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::FlipKernel<paddle::platform::CUDADeviceContext, bool>);
