#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/lookup_table_v2_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T, int BlockDimX, int BlockDimY, int GridDimX,
          bool PaddingFlag>
__global__ void LookupTableV2(T *output, const T *table, const int64_t *ids,
                              const int64_t N, const int64_t K, const int64_t D,
                              const int64_t padding_idx) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int64_t id = ids[idy];
    PADDLE_ENFORCE(
        id >= 0,
        "Variable value (input) of OP(fluid.layers.embedding) "
        "expected >= 0 and < %ld, but got %ld. Please check input value.",
        N, id);
    PADDLE_ENFORCE(
        id < N,
        "Variable value (input) of OP(fluid.layers.embedding) "
        "expected >= 0 and < %ld, but got %ld. Please check input value.",
        N, id);
    T *out = output + idy * D;
    const T *tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      if (PaddingFlag) {
        if (id == padding_idx)
          out[i] = static_cast<T>(0);
        else
          out[i] = tab[i];
      } else {
        out[i] = tab[i];
      }
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void LookupTableV2Grad(T *table, const T *output, const int64_t *ids,
                                  const int64_t N, const int64_t K,
                                  const int64_t D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int64_t id = ids[idy];
    PADDLE_ENFORCE(
        id >= 0,
        "Variable value (input) of OP(fluid.layers.embedding) "
        "expected >= 0 and < %ld, but got %ld. Please check input value.",
        N, id);
    PADDLE_ENFORCE(
        id < N,
        "Variable value (input) of OP(fluid.layers.embedding) "
        "expected >= 0 and < %ld, but got %ld. Please check input value.",
        N, id);
    const T *out = output + idy * D;
    T *tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      paddle::platform::CudaAtomicAdd(&tab[i], out[i]);
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T>
class LookupTableV2CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *table_t = context.Input<LoDTensor>("W");
    auto *ids_t = context.Input<LoDTensor>("Ids");
    auto *output_t = context.Output<LoDTensor>("Out");
    int64_t padding_idx = context.Attr<int64_t>("padding_idx");

    auto id_name = context.Inputs("Ids").front();
    auto out_name = context.Outputs("Out").front();

    size_t N = table_t->dims()[0];
    size_t D = table_t->dims()[1];
    size_t K = ids_t->numel();

    auto *ids = ids_t->data<int64_t>();
    auto *table = table_t->data<T>();
    auto *output = output_t->mutable_data<T>(context.GetPlace());

    dim3 threads(128, 8);
    dim3 grids(8, 1);

    if (padding_idx == -1)
      LookupTableV2<
          T, 128, 8, 8,
          false><<<grids, threads, 0, context.cuda_device_context().stream()>>>(
          output, table, ids, N, K, D, padding_idx);
    else
      LookupTableV2<
          T, 128, 8, 8,
          true><<<grids, threads, 0, context.cuda_device_context().stream()>>>(
          output, table, ids, N, K, D, padding_idx);
  }
};

template <typename T>
class LookupTableV2GradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto &dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    bool is_sparse = context.Attr<bool>("is_sparse");

    // Since paddings are not trainable and fixed in forward, the gradient of
    // paddings makes no sense and we don't deal with it in backward.
    if (is_sparse) {
      auto *ids = context.Input<LoDTensor>("Ids");
      auto *table = context.Input<LoDTensor>("W");
      auto *d_output = context.Input<LoDTensor>(framework::GradVarName("Out"));
      auto *d_table = context.Output<SelectedRows>(framework::GradVarName("W"));

      auto *ids_data = ids->data<int64_t>();
      int64_t ids_num = ids->numel();

      auto stream = dev_ctx.stream();
      // copy GPU memory to CPU pinned memory
      framework::Vector<int64_t> new_rows;
      new_rows.resize(ids_num);
      auto gpu_place = boost::get<platform::CUDAPlace>(context.GetPlace());

      // TODO(yuyang18): Strange code here.
      memory::Copy(gpu_place, new_rows.CUDAMutableData(context.GetPlace()),
                   gpu_place, ids_data, ids_num * sizeof(int64_t), stream);
      d_table->set_rows(new_rows);

      auto *d_table_value = d_table->mutable_value();
      d_table_value->Resize({ids_num, table->dims()[1]});
      d_table_value->mutable_data<T>(context.GetPlace());

      auto *d_table_data = d_table_value->data<T>();
      auto *d_output_data = d_output->data<T>();
      auto d_output_dims = d_output->dims();
      PADDLE_ENFORCE_EQ(
          d_table_value->dims(),
          framework::flatten_to_2d(d_output_dims, d_output_dims.size() - 1));
      memory::Copy(gpu_place, d_table_data, gpu_place, d_output_data,
                   d_output->numel() * sizeof(T), stream);

    } else {
      auto ids_t = context.Input<LoDTensor>("Ids");
      auto d_output_t = context.Input<LoDTensor>(framework::GradVarName("Out"));
      auto d_table_t = context.Output<LoDTensor>(framework::GradVarName("W"));

      int N = d_table_t->dims()[0];
      int D = d_table_t->dims()[1];
      int K = ids_t->numel();
      const int64_t *ids = ids_t->data<int64_t>();
      const T *d_output = d_output_t->data<T>();
      T *d_table = d_table_t->mutable_data<T>(context.GetPlace());

      auto t = framework::EigenVector<T>::Flatten(*d_table_t);
      t.device(*dev_ctx.eigen_device()) = t.constant(static_cast<T>(0));

      dim3 threads(128, 8);
      dim3 grids(8, 1);
      LookupTableV2Grad<T, 128, 8, 8><<<grids, threads, 0, dev_ctx.stream()>>>(
          d_table, d_output, ids, N, K, D);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(lookup_table_v2, ops::LookupTableV2CUDAKernel<float>,
                        ops::LookupTableV2CUDAKernel<double>,
                        ops::LookupTableV2CUDAKernel<plat::float16>);
REGISTER_OP_CUDA_KERNEL(lookup_table_v2_grad,
                        ops::LookupTableV2GradCUDAKernel<float>,
                        ops::LookupTableV2GradCUDAKernel<double>,
                        ops::LookupTableV2GradCUDAKernel<plat::float16>);
