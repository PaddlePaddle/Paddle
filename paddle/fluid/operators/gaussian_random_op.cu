/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <thrust/random.h>

#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/phi/kernels/funcs/index_impl.cu.h"

namespace paddle {
namespace operators {

template <typename T>
struct GaussianGenerator {
  T mean_, std_;
  unsigned int seed_;
  unsigned int offset_ = 0;

  __host__ __device__ GaussianGenerator(T mean, T std, int seed)
      : mean_(mean), std_(std), seed_(seed) {}

  __host__ __device__ GaussianGenerator(T mean, T std, int seed, int offset)
      : mean_(mean), std_(std), seed_(seed), offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    using MT = typename details::MPTypeTrait<T>::Type;
    thrust::normal_distribution<MT> dist(static_cast<MT>(mean_),
                                         static_cast<MT>(std_));
    unsigned int new_n = n + offset_;
    rng.discard(new_n);
    MT out = dist(rng);
    return static_cast<T>(out);
  }
};

template <typename T>
class GPUGaussianRandomBatchSizeLikeKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* tensor = context.Output<framework::Tensor>("Out");
    T* data = tensor->mutable_data<T>(context.GetPlace());
    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    T mean = static_cast<T>(context.Attr<float>("mean"));
    T std = static_cast<T>(context.Attr<float>("std"));
    int64_t size = tensor->numel();

    int device_id = context.GetPlace().GetDeviceId();
    auto gen_cuda = framework::DefaultCUDAGenerator(device_id);
    auto& dev_cxt = context.template device_context<phi::GPUContext>();

    if (seed == 0) {
      // use global Generator seed
      auto seed_offset = gen_cuda->IncrementOffset(1);
      uint64_t seed = seed_offset.first;
      uint64_t offset = seed_offset.second;
      auto func = GaussianGenerator<T>(mean, std, seed, size * offset);
      phi::IndexKernel<T, GaussianGenerator<T>>(dev_cxt, tensor, func);
    } else {
      auto func = GaussianGenerator<T>(mean, std, seed);
      phi::IndexKernel<T, GaussianGenerator<T>>(dev_cxt, tensor, func);
    }
  }
};
}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    gaussian_random_batch_size_like,
    paddle::operators::GPUGaussianRandomBatchSizeLikeKernel<
        paddle::platform::float16>,
    paddle::operators::GPUGaussianRandomBatchSizeLikeKernel<float>,
    paddle::operators::GPUGaussianRandomBatchSizeLikeKernel<double>);
