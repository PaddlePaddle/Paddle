#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifndef _USE_MATH_DEFINES
#define _USE_MATH_DEFINES
#endif
#include "paddle/fluid/operators/eigen/eigen_function.h"
#include "paddle/fluid/platform/eigen_ext.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T>
struct EigenErf<Eigen::GpuDevice, T> {
  using InType = Eigen::TensorMap<
      Eigen::Tensor<const T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  using OutType =
      Eigen::TensorMap<Eigen::Tensor<T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  static void Eval(const Eigen::GpuDevice& dev, OutType out, const InType& in) {
    out.device(dev) = in.erf();
  }
};

template <typename T>
struct EigenErfGrad<Eigen::GpuDevice, T> {
  using InType = Eigen::TensorMap<
      Eigen::Tensor<const T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  using OutType =
      Eigen::TensorMap<Eigen::Tensor<T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  static void Eval(const Eigen::GpuDevice& dev, OutType din, const InType& in,
                   const InType& dout) {
    din.device(dev) =
        dout * static_cast<T>(M_2_SQRTPI) * (-(in.square())).exp();
  }
};

#define INSTANTIATION(FUNCTOR)                       \
  template struct FUNCTOR<Eigen::GpuDevice, float>;  \
  template struct FUNCTOR<Eigen::GpuDevice, double>; \
  template struct FUNCTOR<Eigen::GpuDevice, platform::float16>
INSTANTIATION(EigenErf);
INSTANTIATION(EigenErfGrad);
#undef INSTANTIATION

}  // namespace operators
}  // namespace paddle
