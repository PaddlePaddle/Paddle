#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/rank_attention.cu.h"
#include "paddle/fluid/operators/rank_attention_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using framework::Tensor;

template <typename DeviceContext, typename T>
class RankAttentionCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *X = ctx.Input<Tensor>("X");
    auto *rank_offset = ctx.Input<Tensor>("RankOffset");
    auto *param = ctx.Input<Tensor>("RankParam");
    auto *input_help = ctx.Output<Tensor>("InputHelp");
    auto *ins_rank = ctx.Output<Tensor>("InsRank");
    int max_rank = ctx.Attr<int>("MaxRank");
    int64_t max_size = ctx.Attr<int>("MaxSize");
    auto *Out = ctx.Output<Tensor>("Out");

    // check dims
    auto x_dims = X->dims();
    auto ins_num = x_dims[0];
    auto x_fea_dim = x_dims[1];
    auto para_dims = param->dims();
    auto para_row = para_dims[0];
    auto para_col = para_dims[1];
    auto rank_offset_dims = rank_offset->dims();
    PADDLE_ENFORCE_EQ(
        rank_offset_dims[0], ins_num,
        platform::errors::InvalidArgument("Input(RankOffset) has wrong rows."));
    PADDLE_ENFORCE_EQ((rank_offset_dims[1] - 1) / 2, max_rank,
                      platform::errors::InvalidArgument(
                          "Input(RankOffset) has wrong columns."));
    PADDLE_ENFORCE_EQ(
        max_rank * max_rank * x_fea_dim, para_row,
        platform::errors::InvalidArgument("Input(RankParam) has wrong rows."));

    int block_matrix_row = max_rank * x_fea_dim;

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    int max_ins = std::max(ins_num, max_size);

    Tensor param_help;
    param_help = ctx.AllocateTmpTensor<T, DeviceContext>(
        {max_ins * block_matrix_row, para_col}, dev_ctx);
    param_help.mutable_data<T>(ctx.GetPlace());

    input_help->Resize({max_ins, block_matrix_row});
    ins_rank->Resize({max_ins, 1});
    input_help->mutable_data<T>(ctx.GetPlace());
    ins_rank->mutable_data<T>(ctx.GetPlace());
    Out->mutable_data<T>(ctx.GetPlace());

    // initialize
    auto param_help_eigen = framework::EigenVector<T>::Flatten(param_help);
    auto input_help_eigen = framework::EigenVector<T>::Flatten(*input_help);
    auto ins_rank_eigen = framework::EigenVector<T>::Flatten(*ins_rank);
    auto out_eigen = framework::EigenVector<T>::Flatten(*Out);

    auto &place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();

    param_help_eigen.device(place) =
        param_help_eigen.constant(static_cast<T>(0));
    input_help_eigen.device(place) =
        input_help_eigen.constant(static_cast<T>(0));
    ins_rank_eigen.device(place) = ins_rank_eigen.constant(static_cast<T>(-1));
    out_eigen.device(place) = out_eigen.constant(static_cast<T>(0));

    // get data ptr
    T *input_help_data = input_help->data<T>();
    T *param_help_data = param_help.data<T>();
    T *ins_rank_data = ins_rank->data<T>();
    T *out_data = Out->data<T>();

    expand_rank_attention_input(
        ctx.cuda_device_context().stream(), X->data<T>(), ins_num, x_fea_dim,
        input_help_data, ins_num, block_matrix_row, rank_offset->data<int>(),
        rank_offset_dims[0], rank_offset_dims[1], ins_rank_data, max_rank);

    expand_rank_attention_param(
        ctx.cuda_device_context().stream(), X->data<T>(), ins_num, x_fea_dim,
        rank_offset->data<int>(), rank_offset_dims[0], rank_offset_dims[1],
        param->data<T>(), para_row, para_col, param_help_data,
        ins_num * block_matrix_row, para_col, max_rank);

    CBLAS_TRANSPOSE transA = CblasNoTrans;
    CBLAS_TRANSPOSE transB = CblasNoTrans;

    T alpha = 1;
    T beta = 0;
    int64_t strideA = block_matrix_row;
    int64_t strideB = block_matrix_row * para_col;

    auto blas = math::GetBlas<platform::CUDADeviceContext, T>(dev_ctx);
    blas.BatchedGEMM(transA, transB, 1, para_col, block_matrix_row, alpha,
                     input_help_data, param_help_data, beta, out_data, ins_num,
                     strideA, strideB);
  }
};

template <typename DeviceContext, typename T>
class RankAttentionGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *X = ctx.Input<Tensor>("X");                     // not use data
    auto *rank_offset = ctx.Input<Tensor>("RankOffset");  // not use data
    auto *param = ctx.Input<Tensor>("RankParam");         // not use data
    auto *input_help = ctx.Input<Tensor>("InputHelp");
    auto *ins_rank = ctx.Input<Tensor>("InsRank");
    auto *dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    int64_t max_size = ctx.Attr<int>("MaxSize");

    auto *drank_para = ctx.Output<Tensor>(framework::GradVarName("RankParam"));

    // get dim
    auto x_dims = X->dims();
    auto ins_num = x_dims[0];
    auto x_fea_dim = x_dims[1];
    auto para_dims = param->dims();
    auto para_row = para_dims[0];
    auto para_col = para_dims[1];
    auto rank_offset_dims = rank_offset->dims();
    auto max_rank = (rank_offset_dims[1] - 1) / 2;
    int block_matrix_row = max_rank * x_fea_dim;
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto &place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();

    int max_ins = std::max(ins_num, max_size);
    // initialize out grad
    drank_para->mutable_data<T>(ctx.GetPlace());
    auto drank_para_eigen = framework::EigenVector<T>::Flatten(*drank_para);
    drank_para_eigen.device(place) =
        drank_para_eigen.constant(static_cast<T>(0));

    // copy data
    Tensor param_grad;
    param_grad = ctx.AllocateTmpTensor<T, DeviceContext>(
        {max_ins * block_matrix_row, para_col}, dev_ctx);
    param_grad.mutable_data<T>(ctx.GetPlace());
    // initialize
    auto param_grad_eigen = framework::EigenVector<T>::Flatten(param_grad);
    param_grad_eigen.device(place) =
        param_grad_eigen.constant(static_cast<T>(0));
    // get data ptr
    const T *input_help_data = input_help->data<T>();
    const T *ins_rank_data = ins_rank->data<T>();
    T *param_grad_data = param_grad.data<T>();

    auto blas = math::GetBlas<platform::CUDADeviceContext, T>(dev_ctx);
    T alpha = 1;
    T beta = 0;

    // get param_grad
    CBLAS_TRANSPOSE transA = CblasTrans;
    CBLAS_TRANSPOSE transB = CblasNoTrans;
    int64_t strideA = block_matrix_row;
    int64_t strideB = para_col;
    blas.BatchedGEMM(transA, transB, block_matrix_row, para_col, 1, alpha,
                     input_help_data, dout->data<T>(), beta, param_grad_data,
                     ins_num, strideA, strideB);
    // merge param_grad to get drank_para
    merge_rank_attention_param_grad(
        ctx.cuda_device_context().stream(), param_grad_data,
        ins_num * block_matrix_row, para_col, drank_para->data<T>(), para_row,
        para_col, ins_rank_data, ins_num, max_rank, x_fea_dim);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using GPUCtx = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(rank_attention,
                        ops::RankAttentionCUDAKernel<GPUCtx, float>,
                        ops::RankAttentionCUDAKernel<GPUCtx, double>);

REGISTER_OP_CUDA_KERNEL(rank_attention_grad,
                        ops::RankAttentionGradOpCUDAKernel<GPUCtx, float>,
                        ops::RankAttentionGradOpCUDAKernel<GPUCtx, double>);
