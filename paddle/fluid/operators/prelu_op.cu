#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include <vector>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/prelu.h"
#include "paddle/fluid/operators/prelu_op.h"
#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

#define CUDA_NUM_THREADS 1024

inline static int PADDLE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename DeviceContext, typename T>
class CUDAPReluKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* out = context.Output<Tensor>("Out");

    const T* x_ptr = x->data<T>();
    T* o_ptr = out->mutable_data<T>(context.GetPlace());

    const T* alpha_ptr = alpha->data<T>();
    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();

    VLOG(4) << "dim[0]:" << dim[0] << ", dim[1]:" << dim[1]
            << ", numel:" << numel;

    if (mode == "channel") {
      math::PreluChannelWiseDirectCUDAFunctor<T> prelu_channel_wise;
      prelu_channel_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, dim[0], dim[1], numel);
    } else if (mode == "element") {
      math::PreluElementWiseDirectCUDAFunctor<T> prelu_element_wise;
      prelu_element_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, dim[0], numel);
    } else {
      math::PreluScalarDirectCUDAFunctor<T> prelu_scalar;
      prelu_scalar(context.cuda_device_context().stream(), x_ptr, alpha_ptr,
                   o_ptr, numel);
    }
  }
};

enum PRELU_MODE { Element, Channel, Scalar };

template <typename T>
__global__ void PReluOpGradKernel(const T* x_ptr, const T* alpha_ptr,
                                  const T* dy_ptr, T* dx_ptr, T* dalpha_ptr,
                                  size_t channel_num, size_t plane_size,
                                  size_t spatial_size, size_t numel,
                                  PRELU_MODE mode) {
  CUDA_KERNEL_LOOP(index, numel) {
    T scale;
    if (mode == Element) {
      size_t element_index = index % spatial_size;
      scale = alpha_ptr[element_index];
    } else if (mode == Channel) {
      size_t temp = index / plane_size;
      size_t channel_index = temp % channel_num;
      scale = alpha_ptr[channel_index];
    } else {
      scale = alpha_ptr[0];
    }
    T x = x_ptr[index];
    T dy = dy_ptr[index];
    if (dx_ptr != nullptr) dx_ptr[index] = (x > 0) ? dy : scale * dy;
    if (dalpha_ptr != nullptr) dalpha_ptr[index] = (x > 0) ? 0 : x * dy;
  }
}

template <typename T>
class PreluOpGradFunctor {
 public:
  void operator()(gpuStream_t stream, const T* x, const T* alpha, const T* dy,
                  T* dx, T* dalpha, const framework::DDim& input_dims,
                  PRELU_MODE mode) {
    size_t numel = 1;
    for (size_t i = 0; i < input_dims.size(); ++i) {
      numel *= input_dims[i];
    }
    size_t plane_size = numel / input_dims[0] / input_dims[1];
    size_t spatial_size = numel / input_dims[0];

    PReluOpGradKernel<
        T><<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
        x, alpha, dy, dx, dalpha, input_dims[1], plane_size, spatial_size,
        numel, mode);
  }
};

template <typename T>
struct IdentityFunctor {
  HOSTDEVICE inline T operator()(const T& x) const { return x; }
};

template <typename DeviceContext, typename T>
class CUDAPReluGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* dx = context.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* dalpha = context.Output<Tensor>(framework::GradVarName("Alpha"));

    const T* x_ptr = x->data<T>();
    const T* alpha_ptr = alpha->data<T>();
    const T* dy_ptr = dy->data<T>();
    T* dx_ptr = dx ? dx->mutable_data<T>(context.GetPlace()) : nullptr;
    T* dalpha_ptr =
        dalpha ? dalpha->mutable_data<T>(context.GetPlace()) : nullptr;

    if (!dx && !dalpha) return;

    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();
    std::vector<int> input_shape = framework::vectorize<int>(dim);
    auto stream = context.cuda_device_context().stream();

    T* dalpha_tmp_ptr;
    Tensor dalpha_tmp;
    if (dalpha_ptr == nullptr) {
      dalpha_tmp_ptr = dalpha_ptr;
    } else {
      auto& dev_ctx = context.template device_context<DeviceContext>();
      dalpha_tmp = context.AllocateTmpTensor<T, DeviceContext>(dim, dev_ctx);
      dalpha_tmp_ptr = dalpha_tmp.mutable_data<T>(context.GetPlace());
    }

    PRELU_MODE m;
    if (mode == "element") {
      m = Element;
    } else if (mode == "channel") {
      m = Channel;
    } else {
      m = Scalar;
    }
    PreluOpGradFunctor<T> prelu_grad;
    prelu_grad(stream, x_ptr, alpha_ptr, dy_ptr, dx_ptr, dalpha_tmp_ptr, dim,
               m);

    if (dalpha_tmp_ptr == nullptr) return;

    std::vector<int> reduce_dims;
    for (size_t i = 0; i < dim.size(); i++) {
      if (mode == "channel" && i == 1) continue;
      if (mode == "element" && i != 0) continue;
      reduce_dims.push_back(i);
    }

#ifdef __HIPCC__
    TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
        dalpha_tmp, dalpha, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        IdentityFunctor<T>(), stream);
#else
    TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
        dalpha_tmp, dalpha, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        IdentityFunctor<T>(), stream);
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    prelu, ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    prelu_grad,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, double>);
