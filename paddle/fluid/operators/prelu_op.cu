#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include <vector>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/prelu.h"
#include "paddle/fluid/operators/prelu_op.h"
#include "paddle/fluid/operators/reduce_ops/reduce_op.cu.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

#define CUDA_NUM_THREADS 1024

inline static int PADDLE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename DeviceContext, typename T>
class CUDAPReluKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* out = context.Output<Tensor>("Out");

    const T* x_ptr = x->data<T>();
    T* o_ptr = out->mutable_data<T>(context.GetPlace());

    const T* alpha_ptr = alpha->data<T>();
    auto& mode = context.Attr<std::string>("mode");
    auto& data_format = context.Attr<std::string>("data_format");

    int numel = x->numel();
    auto dim = x->dims();
    auto x_rank = dim.size();

    VLOG(4) << "dim[0]:" << dim[0] << ", dim[1]:" << dim[1] << ", dim["
            << x_rank - 1 << "]:" << dim[x_rank - 1] << ", numel:" << numel;

    if (mode == "channel") {
      bool channel_last = data_format == "NHWC";
      size_t channel = channel_last ? dim[x_rank - 1] : dim[1];
      math::PreluChannelWiseDirectCUDAFunctor<T> prelu_channel_wise;
      prelu_channel_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, dim[0], channel, channel_last,
                         numel);
    } else if (mode == "element") {
      math::PreluElementWiseDirectCUDAFunctor<T> prelu_element_wise;
      prelu_element_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, dim[0], numel);
    } else {
      math::PreluScalarDirectCUDAFunctor<T> prelu_scalar;
      prelu_scalar(context.cuda_device_context().stream(), x_ptr, alpha_ptr,
                   o_ptr, numel);
    }
  }
};

enum PRELU_MODE { Element, ChannelFirst, ChannelLast, Scalar };

template <typename T>
__global__ void PReluOpGradKernel(const T* x_ptr, const T* alpha_ptr,
                                  const T* dy_ptr, T* dx_ptr, T* dalpha_ptr,
                                  size_t channel_num, size_t plane_size,
                                  size_t spatial_size, size_t numel,
                                  PRELU_MODE mode) {
  CUDA_KERNEL_LOOP(index, numel) {
    T scale;
    if (mode == Element) {
      size_t element_index = index % spatial_size;
      scale = alpha_ptr[element_index];
    } else if (mode == ChannelFirst) {
      size_t temp = index / plane_size;
      size_t channel_index = temp % channel_num;
      scale = alpha_ptr[channel_index];
    } else if (mode == ChannelLast) {
      size_t channel_index = index % channel_num;
      scale = alpha_ptr[channel_index];
    } else {
      scale = alpha_ptr[0];
    }
    T x = x_ptr[index];
    T dy = dy_ptr[index];
    T zero = static_cast<T>(0);
    if (dx_ptr != nullptr) dx_ptr[index] = (x > zero) ? dy : scale * dy;
    if (dalpha_ptr != nullptr) dalpha_ptr[index] = (x > zero) ? zero : x * dy;
  }
}

template <typename T>
class PreluOpGradFunctor {
 public:
  void operator()(gpuStream_t stream, const T* x, const T* alpha, const T* dy,
                  T* dx, T* dalpha, const framework::DDim& input_dims,
                  PRELU_MODE mode) {
    size_t numel = 1;
    for (size_t i = 0; i < input_dims.size(); ++i) {
      numel *= input_dims[i];
    }
    size_t plane_size = numel / input_dims[0] / input_dims[1];
    size_t spatial_size = numel / input_dims[0];
    size_t channel =
        mode == ChannelLast ? input_dims[input_dims.size() - 1] : input_dims[1];

    PReluOpGradKernel<
        T><<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
        x, alpha, dy, dx, dalpha, channel, plane_size, spatial_size, numel,
        mode);
  }
};

template <typename DeviceContext, typename T>
class CUDAPReluGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* dx = context.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* dalpha = context.Output<Tensor>(framework::GradVarName("Alpha"));

    const T* x_ptr = x->data<T>();
    const T* alpha_ptr = alpha->data<T>();
    const T* dy_ptr = dy->data<T>();
    T* dx_ptr = dx ? dx->mutable_data<T>(context.GetPlace()) : nullptr;
    T* dalpha_ptr =
        dalpha ? dalpha->mutable_data<T>(context.GetPlace()) : nullptr;

    if (!dx && !dalpha) return;

    auto& mode = context.Attr<std::string>("mode");
    auto& data_format = context.Attr<std::string>("data_format");

    int numel = x->numel();
    auto dim = x->dims();
    auto x_rank = dim.size();
    std::vector<int> input_shape = phi::vectorize<int>(dim);
    auto stream = context.cuda_device_context().stream();

    T* dalpha_tmp_ptr;
    Tensor dalpha_tmp;
    if (dalpha_ptr == nullptr) {
      dalpha_tmp_ptr = dalpha_ptr;
    } else {
      auto& dev_ctx = context.template device_context<DeviceContext>();
      dalpha_tmp = context.AllocateTmpTensor<T, DeviceContext>(dim, dev_ctx);
      dalpha_tmp_ptr = dalpha_tmp.mutable_data<T>(context.GetPlace());
    }

    PRELU_MODE m;
    bool channel_last = false;
    if (mode == "element") {
      m = Element;
    } else if (mode == "channel") {
      channel_last = data_format == "NHWC";
      m = channel_last ? ChannelLast : ChannelFirst;
    } else {
      m = Scalar;
    }
    PreluOpGradFunctor<T> prelu_grad;
    prelu_grad(stream, x_ptr, alpha_ptr, dy_ptr, dx_ptr, dalpha_tmp_ptr, dim,
               m);

    if (dalpha_tmp_ptr == nullptr) return;

    std::vector<int> reduce_dims;
    for (size_t i = 0; i < dim.size(); i++) {
      if (mode == "channel" && !channel_last && i == 1) continue;
      if (mode == "channel" && channel_last && i == dim.size() - 1) continue;
      if (mode == "element" && i != 0) continue;
      reduce_dims.push_back(i);
    }

    TensorReduceImpl<T, T, kps::AddFunctor, kps::IdentityFunctor<T>>(
        context.cuda_device_context(), dalpha_tmp, dalpha,
        kps::IdentityFunctor<T>(), reduce_dims, stream);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    prelu, ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, plat::float16>,
    ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    prelu_grad,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext,
                             plat::float16>,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, double>);
