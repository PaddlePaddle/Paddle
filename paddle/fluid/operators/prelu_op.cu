#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include <vector>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/prelu.h"
#include "paddle/fluid/operators/prelu_op.h"
#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

#define CUDA_NUM_THREADS 1024

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

inline static int PADDLE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename DeviceContext, typename T>
class CUDAPReluKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* out = context.Output<Tensor>("Out");

    const T* x_ptr = x->data<T>();
    T* o_ptr = out->mutable_data<T>(context.GetPlace());

    const T* alpha_ptr = alpha->data<T>();
    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();
    std::vector<int> input_shape = framework::vectorize<int>(dim);

    if (mode == "channel") {
      math::PreluChannelWiseDirectCUDAFunctor<T> prelu_channel_wise;
      prelu_channel_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, input_shape);
    } else if (mode == "element") {
      math::PreluElementWiseDirectCUDAFunctor<T> prelu_element_wise;
      prelu_element_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, input_shape);
    } else {
      math::PreluScalarDirectCUDAFunctor<T> prelu_scalar;
      prelu_scalar(context.cuda_device_context().stream(), x_ptr, alpha_ptr,
                   o_ptr, input_shape);
    }
  }
};

enum PRELU_MODE { Element, Channel, Scalar };

template <typename T>
__global__ void PReluOpGradKernel(const T* x_ptr, const T* alpha_ptr,
                                  const T* dy_ptr, T* dx_ptr, T* dalpha_ptr,
                                  size_t channel_num, size_t plane_size,
                                  size_t spatial_size, size_t numel,
                                  PRELU_MODE mode) {
  size_t index;
  CUDA_KERNEL_LOOP(index, numel) {
    T scale;
    if (mode == Element) {
      size_t element_index = index % spatial_size;
      scale = alpha_ptr[element_index];
    } else if (mode == Channel) {
      size_t temp = index / plane_size;
      size_t channel_index = temp % channel_num;
      scale = alpha_ptr[channel_index];
    } else {
      scale = alpha_ptr[0];
    }
    T x = x_ptr[index];
    T dy = dy_ptr[index];
    if (dx_ptr != nullptr) dx_ptr[index] = (x > 0) ? dy : scale * dy;
    if (dalpha_ptr != nullptr) dalpha_ptr[index] = (x > 0) ? 0 : x * dy;
  }
}

template <typename T>
class PreluOpGradFunctor {
 public:
  void operator()(hipStream_t stream, const T* x, const T* alpha, const T* dy,
                  T* dx, T* dalpha, std::vector<int> input_shape,
                  PRELU_MODE mode) {
    size_t plane_size = input_shape[2] * input_shape[3];
    size_t spatial_size = plane_size * input_shape[1];
    size_t numel = spatial_size * input_shape[0];
    PReluOpGradKernel<
        T><<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
        x, alpha, dy, dx, dalpha, input_shape[1], plane_size, spatial_size,
        numel, mode);
  }
};

template <typename T>
struct IdentityFunctor {
  HOSTDEVICE inline T operator()(const T& x) const { return x; }
};

template <typename DeviceContext, typename T>
class CUDAPReluGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* dx = context.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* dalpha = context.Output<Tensor>(framework::GradVarName("Alpha"));

    const T* x_ptr = x->data<T>();
    const T* alpha_ptr = alpha->data<T>();
    const T* dy_ptr = dy->data<T>();
    T* dx_ptr = dx ? dx->mutable_data<T>(context.GetPlace()) : nullptr;
    T* dalpha_ptr =
        dalpha ? dalpha->mutable_data<T>(context.GetPlace()) : nullptr;

    if (!dx && !dalpha) return;

    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();
    std::vector<int> input_shape = framework::vectorize<int>(dim);
    auto stream = context.cuda_device_context().stream();

    T* dalpha_tmp_ptr;
    Tensor dalpha_tmp;
    if (dalpha_ptr == nullptr) {
      dalpha_tmp_ptr = dalpha_ptr;
    } else {
      auto& dev_ctx = context.template device_context<DeviceContext>();
      dalpha_tmp = context.AllocateTmpTensor<T, DeviceContext>(dim, dev_ctx);
      dalpha_tmp_ptr = dalpha_tmp.mutable_data<T>(context.GetPlace());
    }

    PRELU_MODE m;
    if (mode == "element") {
      m = Element;
    } else if (mode == "channel") {
      m = Channel;
    } else {
      m = Scalar;
    }
    PreluOpGradFunctor<T> prelu_grad;
    prelu_grad(stream, x_ptr, alpha_ptr, dy_ptr, dx_ptr, dalpha_tmp_ptr,
               input_shape, m);

    if (dalpha_tmp_ptr == nullptr) return;

    std::vector<int> reduce_dims;
    for (size_t i = 0; i < input_shape.size(); i++) {
      if (mode == "channel" && i == 1) continue;
      if (mode == "element" && i != 0) continue;
      reduce_dims.push_back(i);
    }

    TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
        dalpha_tmp, dalpha, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        IdentityFunctor<T>(), stream);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    prelu, ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    prelu_grad,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, double>);
