#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <string>
#include <vector>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/prelu.h"
#include "paddle/fluid/operators/prelu_op.h"
#include "paddle/fluid/operators/reduce_ops/cub_reduce.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

static const int CUDA_NUM_THREADS = 1024;
static const int CUDA_MAX_NUM_BLOCKS = 65535;

using Tensor = framework::Tensor;

template <typename DeviceContext, typename T>
class CUDAPReluKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* out = context.Output<Tensor>("Out");

    const T* x_ptr = x->data<T>();
    T* o_ptr = out->mutable_data<T>(context.GetPlace());

    const T* alpha_ptr = alpha->data<T>();
    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();
    std::vector<int> input_shape = framework::vectorize<int>(dim);

    if (mode == "channel") {
      math::PreluChannelWiseDirectCUDAFunctor<T> prelu_channel_wise;
      prelu_channel_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, input_shape);
    } else if (mode == "element") {
      math::PreluElementWiseDirectCUDAFunctor<T> prelu_element_wise;
      prelu_element_wise(context.cuda_device_context().stream(), x_ptr,
                         alpha_ptr, o_ptr, input_shape);
    } else {
      math::PreluScalarDirectCUDAFunctor<T> prelu_scalar;
      prelu_scalar(context.cuda_device_context().stream(), x_ptr, alpha_ptr,
                   o_ptr, input_shape);
    }
  }
};

namespace prelu {
struct ElementWiseMode {};
struct ChannelMode {};
struct ScalarMode {};
} /* namespace prelu */

template <typename T, typename M>
struct AlphaFunctor {
  HOSTDEVICE inline T operator()(const T* alpha, size_t channel,
                                 size_t spatial_size, size_t idx) const {}
};

template <typename T>
struct AlphaFunctor<T, prelu::ElementWiseMode> {
  HOSTDEVICE inline T operator()(const T* alpha, size_t channel,
                                 size_t spatial_size, size_t idx) const {
    return alpha[blockIdx.x * spatial_size + idx];
  }
};

template <typename T>
struct AlphaFunctor<T, prelu::ChannelMode> {
  HOSTDEVICE inline T operator()(const T* alpha, size_t channel,
                                 size_t spatial_size, size_t idx) const {
    return alpha[blockIdx.x % channel];
  }
};

template <typename T>
struct AlphaFunctor<T, prelu::ScalarMode> {
  HOSTDEVICE inline T operator()(const T* alpha, size_t channel,
                                 size_t spatial_size, size_t idx) const {
    return alpha[0];
  }
};

template <typename T, typename M>
__global__ void PReluGradElementWiseKernel(const T* x_ptr, const T* y_ptr,
                                           const T* alpha_ptr, const T* dy_ptr,
                                           T* dx_ptr, T* dalpha_ptr,
                                           size_t channel,
                                           size_t spatial_size) {
  size_t offset = blockIdx.x * spatial_size;
  AlphaFunctor<T, M> alpha_func;

  for (size_t i = threadIdx.x; i < spatial_size; i += blockDim.x) {
    T y = y_ptr[offset + i];
    T x = x_ptr[offset + i];
    T dy = dy_ptr[offset + i];
    T alpha = alpha_func(alpha_ptr, channel, spatial_size, i);
    if (dx_ptr != nullptr) dx_ptr[offset + i] = (y > 0) ? dy : alpha * dy;
    if (dalpha_ptr != nullptr) dalpha_ptr[offset + i] = (x > 0) ? 0 : x * dy;
  }
}

template <typename T, typename M>
class PreluGradElementwiseFunctor {
 public:
  void operator()(hipStream_t stream, const T* x, const T* y, const T* alpha,
                  const T* dy, T* dx, T* dalpha, std::vector<int> input_shape) {
    size_t unroll = input_shape[0] * input_shape[1];
    size_t spatial_size = input_shape[2] * input_shape[3];
    CHECK_LT(unroll, CUDA_MAX_NUM_BLOCKS);
    PReluGradElementWiseKernel<T, M><<<unroll, CUDA_NUM_THREADS, 0, stream>>>(
        x, y, alpha, dy, dx, dalpha, input_shape[1], spatial_size);
  }
};

template <typename T>
struct IdentityFunctor {
  HOSTDEVICE inline T operator()(const T& x) const { return x; }
};

template <typename DeviceContext, typename T>
class CUDAPReluGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<Tensor>("X");
    auto* y = context.Input<Tensor>("Out");
    auto* alpha = context.Input<Tensor>("Alpha");
    auto* dx = context.Output<Tensor>(framework::GradVarName("X"));
    auto* dy = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* dalpha = context.Output<Tensor>(framework::GradVarName("Alpha"));

    const T* x_ptr = x->data<T>();
    const T* y_ptr = y->data<T>();
    const T* alpha_ptr = alpha->data<T>();
    const T* dy_ptr = dy->data<T>();
    T* dx_ptr = dx ? dx->mutable_data<T>(context.GetPlace()) : nullptr;
    T* dalpha_ptr =
        dalpha ? dalpha->mutable_data<T>(context.GetPlace()) : nullptr;

    if (!dx && !dalpha) return;

    auto& mode = context.Attr<std::string>("mode");

    int numel = x->numel();
    auto dim = x->dims();
    std::vector<int> input_shape = framework::vectorize<int>(dim);
    auto stream = context.cuda_device_context().stream();

    T* dalpha_tmp_ptr;
    Tensor dalpha_tmp;
    if (mode == "element" || dalpha_ptr == nullptr) {
      dalpha_tmp_ptr = dalpha_ptr;
    } else {
      auto& dev_ctx = context.template device_context<DeviceContext>();
      dalpha_tmp = context.AllocateTmpTensor<T, DeviceContext>(dim, dev_ctx);
      dalpha_tmp_ptr = dalpha_tmp.mutable_data<T>(context.GetPlace());
    }

    if (mode == "element") {
      PreluGradElementwiseFunctor<T, prelu::ElementWiseMode> prelu_grad;
      prelu_grad(stream, x_ptr, y_ptr, alpha_ptr, dy_ptr, dx_ptr,
                 dalpha_tmp_ptr, input_shape);
    } else if (mode == "channel") {
      PreluGradElementwiseFunctor<T, prelu::ChannelMode> prelu_grad;
      prelu_grad(stream, x_ptr, y_ptr, alpha_ptr, dy_ptr, dx_ptr,
                 dalpha_tmp_ptr, input_shape);
    } else {
      PreluGradElementwiseFunctor<T, prelu::ScalarMode> prelu_grad;
      prelu_grad(stream, x_ptr, y_ptr, alpha_ptr, dy_ptr, dx_ptr,
                 dalpha_tmp_ptr, input_shape);
    }

    if (mode == "element" || dalpha_tmp_ptr == nullptr) return;

    std::vector<int> reduce_dims;
    for (size_t i = 0; i < input_shape.size(); i++) {
      if (mode == "channel" && i == 1) continue;
      reduce_dims.push_back(i);
    }

    TensorReduce<T, T, hipcub::Sum, IdentityFunctor<T>>(
        dalpha_tmp, dalpha, reduce_dims, static_cast<T>(0), hipcub::Sum(),
        IdentityFunctor<T>(), stream);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    prelu, ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    prelu_grad,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, float>,
    ops::CUDAPReluGradKernel<paddle::platform::CUDADeviceContext, double>);
