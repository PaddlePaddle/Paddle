#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <array>
#include "paddle/fluid/framework/conv_search_cache.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/conv_cudnn_op_cache.h"
#include "paddle/fluid/operators/conv_op.h"
#include "paddle/fluid/operators/math/padding.h"
#include "paddle/fluid/platform/cudnn_helper.h"

DECLARE_int64(cudnn_exhaustive_search_times);

namespace paddle {
namespace operators {

#if CUDNN_VERSION >= 7100
using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using ScopedActivationDescriptor = platform::ScopedActivationDescriptor;
using DataLayout = platform::DataLayout;
using framework::AlgorithmsCache;
using framework::ConvSearchCache;

template <typename T>
using ScalingParamType = typename platform::CudnnDataType<T>::ScalingParamType;

template <typename T>
class CUDNNConvFusionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* bias = ctx.Input<Tensor>("Bias");
    auto* residual = ctx.Input<Tensor>("ResidualData");
    auto* output = ctx.Output<Tensor>("Output");
    output->mutable_data<T>(ctx.GetPlace());

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    const std::string activation = ctx.Attr<std::string>("activation");
    int groups = ctx.Attr<int>("groups");
    int64_t user_workspace_size =
        static_cast<size_t>(ctx.Attr<int>("workspace_size_MB"));
    bool exhaustive_search =
        FLAGS_cudnn_exhaustive_search || ctx.Attr<bool>("exhaustive_search");

    const T* filter_data = filter->data<T>();
    const T* bias_data = bias->data<T>();

    const std::string padding_algorithm =
        ctx.Attr<std::string>("padding_algorithm");

    Tensor transformed_input_channel(input->type());
    Tensor transformed_output(output->type());
    transformed_input_channel = *input;
    transformed_output = *output;
    T* output_data = transformed_output.data<T>();

    const T* residual_data = residual ? residual->data<T>() : output_data;

    // update padding and dilation
    auto in_dims = transformed_input_channel.dims();
    auto filter_dims = filter->dims();
    framework::DDim in_data_dims =
        framework::slice_ddim(in_dims, 2, in_dims.size());

    framework::DDim filter_data_dims =
        framework::slice_ddim(filter_dims, 2, filter_dims.size());
    std::vector<int> ksize = framework::vectorize<int>(filter_data_dims);
    UpdatePaddingAndDilation(&paddings, &dilations, padding_algorithm,
                             in_data_dims, strides, ksize);

    int data_dim = strides.size();  // 2d or 3d
    bool is_sys_pad = math::IsSymmetricPadding(paddings, data_dim);

    Tensor transformed_input;
    std::vector<int> padding_common(data_dim, 0);
    if (!is_sys_pad) {
      std::vector<int> padding_diff(data_dim);
      std::vector<int> new_input_shape_vec(data_dim + 2);
      new_input_shape_vec[0] = transformed_input_channel.dims()[0];
      new_input_shape_vec[1] = transformed_input_channel.dims()[1];

      std::vector<int> input_pad(transformed_input_channel.dims().size() * 2,
                                 0);
      for (size_t i = 0; i < data_dim; ++i) {
        padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
        padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
        new_input_shape_vec[i + 2] =
            transformed_input_channel.dims()[i + 2] + padding_diff[i];
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
      framework::DDim new_input_shape(
          framework::make_ddim(new_input_shape_vec));
      transformed_input.Resize(new_input_shape);
      auto& dev_ctx =
          ctx.template device_context<paddle::platform::CUDADeviceContext>();

      transformed_input =
          ctx.AllocateTmpTensor<T, paddle::platform::CUDADeviceContext>(
              new_input_shape, dev_ctx);
      const int rank = transformed_input_channel.dims().size();
      T pad_value(0.0);
      switch (rank) {
        case 4: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 4>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        case 5: {
          math::PadFunction<paddle::platform::CUDADeviceContext, T, 5>(
              ctx, input_pad, transformed_input_channel, pad_value,
              &transformed_input);
        } break;
        default:
          PADDLE_THROW(platform::errors::PermissionDenied(
              "Operator Conv2DFusion expects Input to be a 4-D or 5-D Tensor. "
              "But recieved the actual dimension = %d, shape = [%s].",
              rank, transformed_input_channel.dims()));
      }

    } else {
      transformed_input = transformed_input_channel;
      if (paddings.size() == data_dim) {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[i];
        }
      } else {
        for (size_t i = 0; i < data_dim; ++i) {
          padding_common[i] = paddings[2 * i];
        }
      }
    }

    const T* input_data = transformed_input.data<T>();

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedFilterDescriptor filter_desc;
    ScopedTensorDescriptor bias_desc;
    ScopedConvolutionDescriptor conv_desc;
    ScopedActivationDescriptor act_desc;
    DataLayout layout = DataLayout::kNCHW;
    if (input->dims().size() == 5) {
      layout = DataLayout::kNCDHW;
    }

    hipdnnConvolutionDescriptor_t cudnn_conv_desc =
        conv_desc.descriptor<T>(padding_common, strides, dilations);
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnSetConvolutionGroupCount(cudnn_conv_desc,
                                                         groups));

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize<int>(transformed_input.dims()));
    hipdnnTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
        layout, framework::vectorize<int>(transformed_output.dims()));
    hipdnnFilterDescriptor_t cudnn_filter_desc = filter_desc.descriptor<T>(
        layout, framework::vectorize<int>(filter->dims()));
    // Now only support NCHW
    std::vector<int> bias_dim = {
        1, static_cast<int>(transformed_output.dims()[1]), 1, 1};
    hipdnnTensorDescriptor_t cudnn_bias_desc =
        bias_desc.descriptor<T>(layout, bias_dim);
    hipdnnActivationDescriptor_t cudnn_act_desc =
        act_desc.descriptor<T>(activation);

    // ------------------- cudnn conv workspace ---------------------
    size_t workspace_size_in_bytes;  // final workspace to allocate.
    size_t workspace_size_limit = 0;
    if (FLAGS_conv_workspace_size_limit > 0 || user_workspace_size > 0) {
      int64_t max_user_size =
          std::min(static_cast<int64_t>(FLAGS_conv_workspace_size_limit),
                   user_workspace_size);
      workspace_size_limit = max_user_size * 1024 * 1024;
    }

    // ------------------- cudnn conv algorithm ---------------------
    hipdnnConvolutionFwdAlgo_t algo;
    auto handle = dev_ctx.cudnn_handle();
    auto workspace_handle = dev_ctx.cudnn_workspace_handle();

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetConvolutionMathType(
        cudnn_conv_desc, HIPDNN_DEFAULT_MATH));
#if CUDNN_VERSION >= 11000
    if (!platform::allow_tf32_cudnn) {
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnSetConvolutionMathType(cudnn_conv_desc,
                                                         CUDNN_FMA_MATH));
    }
#endif  // CUDA_VERSION >= 11000

    auto x_dims = framework::vectorize(transformed_input.dims());
    auto f_dims = framework::vectorize(filter->dims());
    if (!exhaustive_search) {
#if CUDNN_VERSION >= 8000
      int perf_count;
      int best_algo_idx = 0;
      size_t tmp_size = 0;
      std::unique_ptr<hipdnnConvolutionFwdAlgoPerf_t[]> perf_results(
          new hipdnnConvolutionFwdAlgoPerf_t[kNUM_CUDNN_FWD_ALGS]);
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::cudnnGetConvolutionForwardAlgorithm_v7(
              handle, cudnn_input_desc, cudnn_filter_desc, cudnn_conv_desc,
              cudnn_output_desc, kNUM_CUDNN_FWD_ALGS, &perf_count,
              perf_results.get()));
      algo = (perf_results.get())[best_algo_idx].algo;
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnGetConvolutionForwardWorkspaceSize(
              handle, cudnn_input_desc, cudnn_filter_desc, cudnn_conv_desc,
              cudnn_output_desc, algo, &workspace_size_in_bytes));
      if (workspace_size_in_bytes > workspace_size_limit)
        workspace_size_limit = workspace_size_in_bytes;
#else
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnGetConvolutionForwardAlgorithm(
              handle, cudnn_input_desc, cudnn_filter_desc, cudnn_conv_desc,
              cudnn_output_desc, HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
              workspace_size_limit, &algo));
      VLOG(3) << "cuDNN forward algo " << algo;
#endif
    } else {
      std::function<hipdnnConvolutionFwdAlgo_t()> search_func =
          [&]() -> hipdnnConvolutionFwdAlgo_t {
        int returned_algo_count;
        std::array<hipdnnConvolutionFwdAlgoPerf_t, kNUM_CUDNN_FWD_ALGS>
            fwd_perf_stat;
        auto cudnn_find_func = [&](void* cudnn_workspace) {
          PADDLE_ENFORCE_CUDA_SUCCESS(
              platform::dynload::hipdnnFindConvolutionForwardAlgorithmEx(
                  handle, cudnn_input_desc, input_data, cudnn_filter_desc,
                  filter_data, cudnn_conv_desc, cudnn_output_desc, output_data,
                  kNUM_CUDNN_FWD_ALGS, &returned_algo_count,
                  fwd_perf_stat.data(), cudnn_workspace, workspace_size_limit));
        };
        workspace_handle.RunFuncSync(cudnn_find_func, workspace_size_limit);
        VLOG(3) << "Perf result: (algo: stat, time, memory)";
        for (int i = 0; i < returned_algo_count; ++i) {
          const auto& stat = fwd_perf_stat[i];
          VLOG(3) << stat.algo << ": " << stat.status << " " << stat.time << " "
                  << stat.memory;
        }
        return fwd_perf_stat[0].algo;
      };
      AlgorithmsCache<hipdnnConvolutionFwdAlgo_t>& algo_cache =
          *(framework::ConvSearchCache::Instance().GetConvFusion());
      int search_times = ctx.Attr<int>("search_times");
      search_times = std::max(
          static_cast<int>(FLAGS_cudnn_exhaustive_search_times), search_times);
      // TODO(dangqingqing): Unify this if-else.
      if (search_times > 0) {
        // The searched algo will be cached by `search_times` times for
        // different input dimension. For other dimensions, select the algo
        // of closest area.
        algo = algo_cache.GetAlgorithm(x_dims[2] * x_dims[3], search_times, 0,
                                       search_func);
      } else {
        auto dtype = platform::CudnnDataType<T>::type;
        algo = algo_cache.GetAlgorithm(x_dims, f_dims, strides, paddings,
                                       dilations, 0, dtype, search_func);
      }
      VLOG(3) << "choose algo " << algo;
    }

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnGetConvolutionForwardWorkspaceSize(
            handle, cudnn_input_desc, cudnn_filter_desc, cudnn_conv_desc,
            cudnn_output_desc, algo, &workspace_size_in_bytes));
    PADDLE_ENFORCE_LE(
        workspace_size_in_bytes, workspace_size_limit,
        platform::errors::InvalidArgument(
            "The actual workspace size to be allocated for cuDNN is expected "
            "to be less than the limit. But recieved: the actual workspace "
            "size = %d, limit = %d.",
            workspace_size_in_bytes, workspace_size_limit));

    if ((activation == "identity") && (!residual)) {
      // Only the HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM algo is
      // enabled with HIPDNN_ACTIVATION_PATHTRU in cuDNN lib.
      // But test in some case, the speed is slower, change to use
      // hipdnnConvolutionForward and hipdnnAddTensor
      // ------------- cudnn conv forward and bias add ---------------------
      ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnConvolutionForward(
            handle, &alpha, cudnn_input_desc, input_data, cudnn_filter_desc,
            filter_data, cudnn_conv_desc, algo, cudnn_workspace,
            workspace_size_in_bytes, &beta, cudnn_output_desc, output_data));
      };
      workspace_handle.RunFunc(cudnn_func, workspace_size_in_bytes);
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnAddTensor(
          handle, &alpha, cudnn_bias_desc, bias_data, &alpha, cudnn_output_desc,
          output_data));
    } else {
      if (activation == "identity") {
        algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
      }
      // ------------------- cudnn conv+bias+act forward --------------------
      ScalingParamType<T> alpha1 = 1.0f;
      ScalingParamType<T> alpha2 = residual ? 1.0f : 0.0f;
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::cudnnConvolutionBiasActivationForward(
                handle, &alpha1, cudnn_input_desc, input_data,
                cudnn_filter_desc, filter_data, cudnn_conv_desc, algo,
                cudnn_workspace, workspace_size_in_bytes, &alpha2,
                cudnn_output_desc, residual_data, cudnn_bias_desc, bias_data,
                cudnn_act_desc, cudnn_output_desc, output_data));
      };
      workspace_handle.RunFunc(cudnn_func, workspace_size_in_bytes);
    }
    std::vector<int> channels = ctx.Attr<std::vector<int>>("split_channels");
    if (channels.size()) {
      auto outs = ctx.MultiOutput<framework::Tensor>("Outputs");
      if (x_dims[0] == 1) {
        // share data with Output
        framework::Tensor t;
        t.ShareDataWith(*output);
        auto y_dims = output->dims();
        t.Resize({y_dims[1], y_dims[2], y_dims[3]});
        int s = 0;
        for (size_t i = 0; i < channels.size(); ++i) {
          int e = s + channels[i];
          outs[i]->ShareDataWith(t.Slice(s, e));
          outs[i]->Resize({x_dims[0], channels[i], y_dims[2], y_dims[3]});
          s = e;
        }
      } else {
        // TODO(qingiqng): do copy when batch size large than 1
        PADDLE_THROW(platform::errors::Unimplemented(
            "Input with batch size greater than 1 is unsupported. The recieved "
            "batch size is %d, Input's shape is [%s].",
            x_dims[0], framework::make_ddim(x_dims)));
      }
    }
  }
};
#endif

}  // namespace operators
}  // namespace paddle

#if CUDNN_VERSION >= 7100
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(conv2d_fusion, ops::CUDNNConvFusionOpKernel<float>,
                        ops::CUDNNConvFusionOpKernel<double>);
#endif
