#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <time.h>

#include <random>
#include <vector>

#include "paddle/fluid/operators/fused/fused_dropout_test.h"
#include "paddle/fluid/operators/fused/fused_residual_dropout_bias.h"

namespace framework = paddle::framework;
namespace platform = paddle::platform;

/**
 * @brief the unittest of fusedresidualdropoutbias
 * 1. random input data
 * 2. add bias, call paddle dropout op, add residual, and get the base result
 * 3. call FusedResidualDropoutBias function get fused result
 * 4. compare ther base result and fused result
 */

template <typename T>
struct TestFusedResidualDropoutBias {
  uint32_t rows;
  uint32_t cols;
  uint64_t seed;
  float dropout_prob;
  bool is_upscale_in_train;
  bool is_test;  // default false,  Set to true for inference only
  bool has_bias = true;
  framework::Tensor src, residual, bias, out, mask;
  framework::Tensor dsrc, dbias;

  std::vector<T> src_vec, residual_vec, bias_vec;
  std::vector<T> correct_out, correct_dsrc, correct_dbias;
  std::vector<uint8_t> correct_mask;

  platform::CUDAPlace place;
  platform::CUDADeviceContext *ctx;

  TestFusedResidualDropoutBias() {
    rows = 32;
    cols = 32;
    seed = 0;
    dropout_prob = 0.0;
    is_upscale_in_train = false;
    is_test = false;
    has_bias = true;
    platform::DeviceContextPool &pool = platform::DeviceContextPool::Instance();
    auto device_ctx = pool.Get(place);
    ctx = reinterpret_cast<platform::CUDADeviceContext *>(device_ctx);
  }

  TestFusedResidualDropoutBias(int rows_, int cols_, uint64_t seed_ = 0,
                               float dropout_prob_ = 0.0,
                               bool is_upscale_in_train_ = false,
                               bool is_test_ = false) {
    rows = rows_;
    cols = cols_;
    seed = seed_;
    dropout_prob = dropout_prob_;
    is_upscale_in_train = is_upscale_in_train_;
    is_test = is_test_;
    has_bias = true;
    platform::DeviceContextPool &pool = platform::DeviceContextPool::Instance();
    auto device_ctx = pool.Get(place);
    ctx = reinterpret_cast<platform::CUDADeviceContext *>(device_ctx);
  }

  ~TestFusedResidualDropoutBias() {}

  void SetUp() {
    const int n = rows * cols;
    correct_out.resize(n);
    correct_mask.resize(n);
    correct_dsrc.resize(n);
    correct_dbias.resize(cols);

    src_vec.resize(n);
    residual_vec.resize(n);
    bias_vec.resize(cols);
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> dis(0.0, 1.0);

    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        src_vec[i * cols + j] = static_cast<T>(dis(random));
        residual_vec[i * cols + j] = static_cast<T>(dis(random));
        if (i == 0) {
          bias_vec[j] = dis(random);
        }
      }
    }

    framework::TensorFromVector<T>(src_vec, *ctx, &src);
    src.Resize({rows, cols});
    framework::TensorFromVector<T>(residual_vec, *ctx, &residual);
    residual.Resize({rows, cols});
    if (has_bias) {
      framework::TensorFromVector<T>(bias_vec, *ctx, &bias);
      bias.Resize({cols});
    }

    {
      out.mutable_data<T>({rows, cols}, place);
      mask.mutable_data<uint8_t>({rows, cols}, place);
      dsrc.mutable_data<T>({rows, cols}, place);

      if (has_bias) {
        dbias.mutable_data<T>({cols}, place);
      }
    }
  }

  void BaseForward() {
    std::vector<T> out1(rows * cols), out2(rows * cols);
    if (has_bias) {
      // add bias
      for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
          out1[i * cols + j] = src_vec[i * cols + j] + bias_vec[j];
        }
      }
      // call dropout
      Dropout<T>(out1, src.dims(), &out2, &correct_mask, *ctx, seed,
                 dropout_prob, is_upscale_in_train, is_test);
    } else {
      Dropout<T>(src_vec, src.dims(), &out2, &correct_mask, *ctx, seed,
                 dropout_prob, is_upscale_in_train, is_test);
    }
    ctx->Wait();
    // add residual
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        correct_out[i * cols + j] =
            residual_vec[i * cols + j] + out2[i * cols + j];
      }
    }
  }

  void BaseBackward() {
    DropoutGrad<T>(&correct_dsrc, src.dims(), correct_out, correct_mask, *ctx,
                   dropout_prob, is_upscale_in_train);
    // calc dbias
    memset(&correct_dbias[0], 0, cols * sizeof(T));
    if (has_bias) {
      ReduceSum<T>(correct_out, &correct_dbias, rows, cols);
    }
  }

  void FusedForward() {
    const int VecSize = MAX_CACHE_BYTES / sizeof(T);
    auto config = paddle::operators::Get1DBlocksAnd2DGrids(
        *ctx, static_cast<uint64_t>(rows), static_cast<uint64_t>(cols),
        VecSize);

    const int increment = ((cols - 1) / (config.thread_per_block.x *
                                         config.block_per_grid.x * VecSize) +
                           1) *
                          VecSize;

    T *bias_ptr = nullptr;
    if (has_bias) {
      bias_ptr = bias.data<T>();
    }
    paddle::operators::LaunchResidualDropoutBias<T, uint8_t>(
        rows, cols, increment, seed, dropout_prob, is_test, is_upscale_in_train,
        src.data<T>(), residual.data<T>(), bias_ptr, mask.data<uint8_t>(),
        out.data<T>(), *ctx);
    ctx->Wait();
  }

  void FusedBackward() {
    if (is_test) {
      return;
    }

    T *bias_ptr = nullptr;
    if (has_bias) {
      bias_ptr = dbias.data<T>();
    }
    paddle::operators::LaunchResidualDropoutBiasGrad<T, uint8_t>(
        out.data<T>(), mask.data<uint8_t>(), dropout_prob, is_upscale_in_train,
        rows, cols, dsrc.data<T>(), bias_ptr, *ctx);
  }

  void Run() {
    SetUp();
    BaseForward();
    FusedForward();
    BaseBackward();
    FusedBackward();
  }

  void CheckOut(const T diff) {
    const int n = rows * cols;
    std::vector<T> _out(n);
    std::vector<uint8_t> _mask(n);
    framework::TensorToVector(out, *ctx, &_out);
    if (!is_test) {
      framework::TensorToVector<uint8_t>(mask, *ctx, &_mask);
    }
    ctx->Wait();

    for (int i = 0; i < n; i++) {
      EXPECT_LT(std::abs(_out[i] - correct_out[i]), diff);
      if (!is_test) EXPECT_EQ(_mask[i], correct_mask[i]);
    }
  }

  void CheckGrad(const T diff) {
    if (is_test) {
      return;
    }

    const int n = rows * cols;

    std::vector<T> _dsrc(n);
    framework::TensorToVector(dsrc, *ctx, &_dsrc);

    for (int i = 0; i < n; i++) {
      EXPECT_LT(std::abs(_dsrc[i] - correct_dsrc[i]), diff);
    }

    if (has_bias) {
      std::vector<T> _dbias(cols);
      framework::TensorToVector(dbias, *ctx, &_dbias);
      ctx->Wait();
      for (int i = 0; i < cols; i++) {
        EXPECT_LT(std::abs(_dbias[i] - correct_dbias[i]), diff);
      }
    }
  }
};

// test the shape and bias
template <typename T>
static void BaseTest(const bool is_fp16 = false) {
  const int rows = 16;
  T default_diff = !is_fp16 ? static_cast<T>(1e-5) : static_cast<T>(1e-1);
  for (auto cols : {16, 17}) {
    for (auto has_bias : {true, false}) {
      TestFusedResidualDropoutBias<T> test(rows, cols);
      test.has_bias = has_bias;
      test.Run();
      test.CheckOut(default_diff);
      test.CheckGrad(default_diff);
    }
  }
}

TEST(FusedDropout, GPUFusedResidualDropoutBias) { BaseTest<float>(); }

TEST(FusedDropout, GPUFusedResidualDropoutBiasDouble) { BaseTest<double>(); }

TEST(FusedDropout, GPUFusedResidualDropoutBiasFp16) {
  BaseTest<platform::float16>(true);
}

TEST(FusedDropout, GPUFusedResidualDropoutBiasIsUpscaleInTrain) {
  const int rows = 16;
  const int cols = 16;
  for (auto is_upscale_in_train : {true, false}) {
    TestFusedResidualDropoutBias<float> test(rows, cols, 0, 1.0,
                                             is_upscale_in_train, false);
    test.Run();
    test.CheckOut(static_cast<float>(1e-5));
    test.CheckGrad(static_cast<float>(1e-5));
  }
}

TEST(FusedDropout, GPUFusedResidualDropoutBiasIsTest) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 0, 0.35, true, true);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedResidualDropoutBiasSeed) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 125, 0.0, false, false);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedResidualDropoutBiasLargeShape) {
  const int rows = 256;
  const int cols = 4096;
  TestFusedResidualDropoutBias<float> test(rows, cols);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-3));
}
