#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <time.h>

#include <random>
#include <vector>

#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/fused/fused_dropout_act_bias.h"
#include "paddle/fluid/operators/fused/fused_dropout_test.h"
#include "paddle/phi/kernels/funcs/functors.h"

namespace framework = paddle::framework;
namespace platform = paddle::platform;
namespace details = paddle::operators::details;

/**
 * @brief the unittest of fused_dropout_act_bias
 * 1. random input data
 * 2. add bias, call activation, call paddle dropout, and get the base result
 * 3. call FusedDropoutActBias function get fused result
 * 4. compare ther base result and fused result
 */

template <typename T, typename Functor, typename GradFunctor>
struct TestFusedDropoutActBias {
  uint32_t rows;
  uint32_t cols;
  uint64_t seed;
  float dropout_prob;
  bool is_upscale_in_train;
  bool is_test;  // default false,  Set to true for inference only
  bool has_bias = true;
  framework::Tensor src, bias, out, mask;
  framework::Tensor dsrc, dbias;

  std::vector<T> src_vec, bias_vec, out_vec, mask_vec;
  std::vector<T> correct_out, correct_dsrc, correct_dbias;
  std::vector<uint8_t> correct_mask;

  platform::CUDAPlace place;
  platform::CUDADeviceContext *ctx;

  TestFusedDropoutActBias() {
    rows = 32;
    cols = 32;
    seed = 0;
    dropout_prob = 0.0;
    is_upscale_in_train = false;
    is_test = false;
    has_bias = true;
    platform::DeviceContextPool &pool = platform::DeviceContextPool::Instance();
    auto devicectx = pool.Get(place);
    ctx = reinterpret_cast<platform::CUDADeviceContext *>(devicectx);
  }

  TestFusedDropoutActBias(int rows_, int cols_, uint64_t seed_ = 0,
                          float dropout_prob_ = 0.0,
                          bool is_upscale_in_train_ = false,
                          bool is_test_ = false) {
    rows = rows_;
    cols = cols_;
    seed = seed_;
    dropout_prob = dropout_prob_;
    is_upscale_in_train = is_upscale_in_train_;
    is_test = is_test_;
    has_bias = true;
    platform::DeviceContextPool &pool = platform::DeviceContextPool::Instance();
    auto devicectx = pool.Get(place);
    ctx = reinterpret_cast<platform::CUDADeviceContext *>(devicectx);
  }

  ~TestFusedDropoutActBias() {}

  void SetUp() {
    const int n = rows * cols;
    correct_out.resize(n);
    correct_mask.resize(n);
    correct_dsrc.resize(n);
    correct_dbias.resize(cols);

    src_vec.resize(n);
    bias_vec.resize(cols);
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> dis(0.0, 1.0);

    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        src_vec[i * cols + j] = static_cast<T>(dis(random));
        if (i == 0) bias_vec[j] = dis(random);
      }
    }

    framework::TensorFromVector<T>(src_vec, *ctx, &src);
    src.Resize({rows, cols});
    if (has_bias) {
      framework::TensorFromVector<T>(bias_vec, *ctx, &bias);
      bias.Resize({cols});
    }

    {
      out.mutable_data<T>({rows, cols}, place);
      mask.mutable_data<uint8_t>({rows, cols}, place);
      dsrc.mutable_data<T>({rows, cols}, place);

      if (has_bias) {
        dbias.mutable_data<T>({cols}, place);
      }
    }
  }

  void BaseForward() {
    std::vector<T> out1(rows * cols);
    Functor act;
    if (has_bias) {
      // add bias and call activation
      for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
          const T tmp = src_vec[i * cols + j] + bias_vec[j];
          out1[i * cols + j] = act(tmp);
        }
      }
      // call dropout
      Dropout<T>(out1, src.dims(), &correct_out, &correct_mask, *ctx, seed,
                 dropout_prob, is_upscale_in_train, is_test);
    } else {
      for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
          const T tmp = src_vec[i * cols + j];
          out1[i * cols + j] = act(tmp);
        }
      }

      Dropout<T>(out1, src.dims(), &correct_out, &correct_mask, *ctx, seed,
                 dropout_prob, is_upscale_in_train, is_test);
    }
    ctx->Wait();
  }

  void BaseBackward() {
    std::vector<T> _out(rows * cols);
    // call dropout_grad
    DropoutGrad<T>(&_out, src.dims(), correct_out, correct_mask, *ctx,
                   dropout_prob, is_upscale_in_train);

    // calculate dbias
    memset(&correct_dbias[0], 0, cols * sizeof(T));
    GradFunctor act_grad;
    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        T args[2];
        args[0] = _out[i * cols + j];
        if (has_bias) {
          args[1] = src_vec[i * cols + j] + bias_vec[j];
        } else {
          args[1] = src_vec[i * cols + j];
        }
        T val = args[0] * act_grad.UseOut(args[1]);
        correct_dsrc[i * cols + j] = val;
      }
    }

    if (has_bias) {
      // reduce_sum: keep the same calculate order as the GPU
      ReduceSum<T>(correct_dsrc, &correct_dbias, rows, cols);
    }
  }

  void FusedForward() {
    const int VecSize = MAX_CACHE_BYTES / sizeof(T);
    auto config = paddle::operators::Get1DBlocksAnd2DGrids(
        *ctx, static_cast<uint64_t>(rows), static_cast<uint64_t>(cols),
        VecSize);
    const int increment = ((cols - 1) / (config.thread_per_block.x *
                                         config.block_per_grid.x * VecSize) +
                           1) *
                          VecSize;

    T *bias_ptr = nullptr;
    if (has_bias) {
      bias_ptr = bias.data<T>();
    }
    Functor act;
    paddle::operators::LaunchDropoutActBias<T, uint8_t, Functor>(
        act, seed, rows, cols, increment, dropout_prob, is_upscale_in_train,
        is_test, src.data<T>(), bias_ptr, out.data<T>(), mask.data<uint8_t>(),
        *ctx);
    ctx->Wait();
  }

  void FusedBackward() {
    if (is_test) return;

    T *bias_ptr = nullptr;
    T *dbias_ptr = nullptr;
    if (has_bias) {
      dbias_ptr = dbias.data<T>();
      bias_ptr = bias.data<T>();
    }
    GradFunctor act_grad;
    paddle::operators::LaunchDropoutActBiasGrad<T, uint8_t, GradFunctor>(
        act_grad, out.data<T>(), mask.data<uint8_t>(), src.data<T>(), bias_ptr,
        dropout_prob, is_upscale_in_train, rows, cols, dsrc.data<T>(),
        dbias_ptr, *ctx);
  }

  void Run() {
    SetUp();
    BaseForward();
    FusedForward();
    BaseBackward();
    FusedBackward();
  }

  void CheckOut(const T diff) {
    const int n = rows * cols;
    std::vector<T> _out(n);
    std::vector<uint8_t> _mask(n);
    framework::TensorToVector(out, *ctx, &_out);
    if (!is_test) {
      framework::TensorToVector<uint8_t>(mask, *ctx, &_mask);
    }
    ctx->Wait();

    for (int i = 0; i < n; i++) {
      EXPECT_LT(std::abs(_out[i] - correct_out[i]), diff);
      if (!is_test) EXPECT_EQ(_mask[i], correct_mask[i]);
    }
  }

  void CheckGrad(const T diff) {
    if (is_test) return;

    const int n = rows * cols;

    std::vector<T> _dsrc(n);
    framework::TensorToVector(dsrc, *ctx, &_dsrc);

    for (int i = 0; i < n; i++) {
      EXPECT_LT(std::abs(_dsrc[i] - correct_dsrc[i]), diff);
    }

    if (has_bias) {
      std::vector<T> _dbias(cols);
      framework::TensorToVector(dbias, *ctx, &_dbias);
      ctx->Wait();
      for (int i = 0; i < cols; i++) {
        EXPECT_LT(std::abs(_dbias[i] - correct_dbias[i]), diff);
      }
    }
  }
};

// test the shape , bias, activation
template <typename T, typename Functor, typename GradFunctor>
static void BaseTest(const bool is_fp16 = false) {
  const int rows = 16;
  std::vector<int> cols_list = {16, 17};
  bool has_bias[2] = {true, false};
  T default_diff = !is_fp16 ? static_cast<T>(1e-5) : static_cast<T>(1e-1);
  for (auto cols : {16, 17}) {
    for (auto has_bias : {true, false}) {
      TestFusedDropoutActBias<T, Functor, GradFunctor> test(rows, cols);
      test.has_bias = has_bias;
      test.Run();
      test.CheckOut(default_diff);
      test.CheckGrad(default_diff);
    }
  }
}

TEST(FusedDropout, GPUFusedDorpoutActBias) {
  BaseTest<float, phi::funcs::ReluFunctor<float>,
           phi::funcs::ReluGradFunctor<float>>();
  BaseTest<float, paddle::operators::GeluFunctor<float>,
           paddle::operators::GeluGradFunctor<float>>();
}
TEST(FusedDropout, GPUFusedDropoutActBiasDouble) {
  BaseTest<double, phi::funcs::ReluFunctor<double>,
           phi::funcs::ReluGradFunctor<double>>();
  BaseTest<double, paddle::operators::GeluFunctor<double>,
           paddle::operators::GeluGradFunctor<double>>();
}

// test fp16, For inference, check_grad is not required. ref: test_dropout_op.py
TEST(FusedDropout, GPUFusedDropoutActBiasFp16) {
  using fp16 = platform::float16;
  BaseTest<fp16, phi::funcs::ReluFunctor<fp16>,
           phi::funcs::ReluGradFunctor<fp16>>(true);
}

TEST(FusedDropout, GPUFusedDropoutActBiasIsUpscaleInTrain) {
  const int rows = 16;
  const int cols = 16;
  for (auto is_upscale_in_train : {true, false}) {
    TestFusedDropoutActBias<float, phi::funcs::ReluFunctor<float>,
                            phi::funcs::ReluGradFunctor<float>>
        test(rows, cols, 0, 1.0, is_upscale_in_train, false);
    test.Run();
    test.CheckOut(static_cast<float>(1e-5));
    test.CheckGrad(static_cast<float>(1e-3));
  }
}

TEST(FusedDropout, GPUFusedDropoutActBiasIsTest) {
  const int rows = 16;
  const int cols = 16;
  TestFusedDropoutActBias<float, phi::funcs::ReluFunctor<float>,
                          phi::funcs::ReluGradFunctor<float>>
      test(rows, cols, 0, 0.35, true, true);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-3));
}

TEST(FusedDropout, GPUFusedDropoutActBiasSeed) {
  const int rows = 16;
  const int cols = 16;
  TestFusedDropoutActBias<float, phi::funcs::ReluFunctor<float>,
                          phi::funcs::ReluGradFunctor<float>>
      test(rows, cols, 125, 0.0, false, false);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-3));
}

TEST(FusedDropout, GPUFusedDropoutActBiasLargeShape) {
  const int rows = 256;
  const int cols = 4096;
  TestFusedDropoutActBias<float, phi::funcs::ReluFunctor<float>,
                          phi::funcs::ReluGradFunctor<float>>
      test(rows, cols);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-3));
}
