#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_fp16.h>
#endif
#ifdef PADDLE_WITH_HIP
#include <hip/hip_fp16.h>
#endif
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"

#include "paddle/fluid/operators/elementwise/elementwise_add_op.h"
#include "paddle/phi/kernels/funcs/math_function.h"

#include "paddle/fluid/operators/fused/attn_gemm.h"
#include "paddle/fluid/operators/fused/fmha_ref.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DDim = framework::DDim;

template <typename T>
struct SigmoidMultiplyFunctor {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType one = static_cast<MPType>(1.0f);

  // sigmoid(x) = 1 / (1 + exp(-x))
  // out = sigmoid(x) * y
  inline HOSTDEVICE T operator()(T x, T y) const {
    MPType x_mp = static_cast<MPType>(x);
    T sigmoid_out = static_cast<T>(one / (one + exp(-x_mp)));
    return sigmoid_out * y;
  }
};

template <typename T>
struct SigmoidMultiplyGradFunctor {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType one = static_cast<MPType>(1.0f);

  // Gradient of Multiply:
  //  dx = dout * y
  //  dy = dout * x
  // Gradient of Sigmoid: dx = dout * out * (1 - out)
  inline HOSTDEVICE phi::Array<T, 2> operator()(const T dout, const T x,
                                                T y) const {
    MPType x_mp = static_cast<MPType>(x);
    T sigmoid_out = static_cast<T>(one / (one + exp(-x_mp)));
    T d_sigmoid_out = dout * y;
    phi::Array<T, 2> outs;
    outs[0] = d_sigmoid_out * sigmoid_out *
              (static_cast<T>(1.0f) - sigmoid_out);  // dx
    outs[1] = dout * sigmoid_out;                    // dy
    return outs;
  }
};

template <typename T>
void ComputeMergedQKVMatmulForward(const framework::ExecutionContext &ctx,
                                   const Tensor *x, Tensor *qkv_out, int m,
                                   int n, int k) {
  auto *qkv_weight = ctx.Input<Tensor>("QKVWeight");

  // qkv_out = GEMM(x, qkv_weight)
  auto qkv_compute =
      AttnMatMul<T>(ctx.cuda_device_context(), false, true, m, n, k, false);
  qkv_compute.ComputeForward(qkv_weight, x, nullptr, qkv_out, nullptr);
}

template <typename T>
Tensor *ComputeMergedQKVMatmulBackward(const framework::ExecutionContext &ctx,
                                       const Tensor *x, const Tensor *d_qkv_out,
                                       Tensor *d_x, int m, int n, int k,
                                       bool use_addto) {
  auto *qkv_weight = ctx.Input<Tensor>("QKVWeight");
  auto *d_qkv_weight = ctx.Output<Tensor>(framework::GradVarName("QKVWeight"));
  d_qkv_weight->mutable_data<T>(ctx.GetPlace());

  // Gradient of GEMM(x, qkv_weight)
  auto qkv_compute =
      AttnMatMul<T>(ctx.cuda_device_context(), false, true, m, n, k, false);
  qkv_compute.ComputeBackward(x, qkv_weight, d_qkv_out, d_x, d_qkv_weight,
                              nullptr, use_addto);
  return d_x;
}

template <typename T>
void ComputeQKVMatmulForward(const framework::ExecutionContext &ctx,
                             const Tensor *query, const Tensor *key,
                             Tensor *q_out, Tensor *k_out, Tensor *v_out,
                             int q_m, int q_n, int q_k, int k_m, int k_n,
                             int k_k) {
  auto *q_weight = ctx.Input<Tensor>("QueryWeight");
  auto *k_weight = ctx.Input<Tensor>("KeyWeight");
  auto *v_weight = ctx.Input<Tensor>("ValueWeight");

  // q_out = GEMM(query, q_weight)
  auto q_compute = AttnMatMul<T>(ctx.cuda_device_context(), false, false, q_m,
                                 q_n, q_k, false);
  q_compute.ComputeForward(q_weight, query, nullptr, q_out, nullptr);

  // k_out = GEMM(key, k_weight)
  auto k_compute = AttnMatMul<T>(ctx.cuda_device_context(), false, false, k_m,
                                 k_n, k_k, false);
  k_compute.ComputeForward(k_weight, key, nullptr, k_out, nullptr);

  // v_out = GEMM(value, v_weight)
  k_compute.ComputeForward(v_weight, key, nullptr, v_out, nullptr);
}

template <typename T>
Tensor *ComputeQKVMatmulBackward(const framework::ExecutionContext &ctx,
                                 const Tensor *query, const Tensor *key,
                                 const Tensor *d_q_out, const Tensor *d_k_out,
                                 const Tensor *d_v_out, Tensor *d_x,
                                 Tensor *d_key, int q_m, int q_n, int q_k,
                                 int k_m, int k_n, int k_k, bool use_addto) {
  auto *q_weight = ctx.Input<Tensor>("QueryWeight");
  auto *d_q_weight = ctx.Output<Tensor>(framework::GradVarName("QueryWeight"));
  auto *k_weight = ctx.Input<Tensor>("KeyWeight");
  auto *d_k_weight = ctx.Output<Tensor>(framework::GradVarName("KeyWeight"));
  auto *v_weight = ctx.Input<Tensor>("ValueWeight");
  auto *d_v_weight = ctx.Output<Tensor>(framework::GradVarName("ValueWeight"));
  d_q_weight->mutable_data<T>(ctx.GetPlace());
  d_k_weight->mutable_data<T>(ctx.GetPlace());
  d_v_weight->mutable_data<T>(ctx.GetPlace());

  // Gradient of GEMM(key, k_weight)
  auto k_compute = AttnMatMul<T>(ctx.cuda_device_context(), false, false, k_m,
                                 k_n, k_k, false);
  k_compute.ComputeBackward(key, k_weight, d_k_out, d_key, d_k_weight, nullptr,
                            use_addto);

  // Gradient of GEMM(value, v_weight)
  k_compute.ComputeBackward(key, v_weight, d_v_out, d_key, d_v_weight, nullptr,
                            use_addto);

  // Gradient of GEMM(query, q_weight)
  auto q_compute = AttnMatMul<T>(ctx.cuda_device_context(), false, false, q_m,
                                 q_n, q_k, false);
  q_compute.ComputeBackward(query, q_weight, d_q_out, d_x, d_q_weight, nullptr,
                            use_addto);
  return d_x;
}

template <typename T>
Tensor *ComputeGatingLinearForward(const framework::ExecutionContext &ctx,
                                   const Tensor *x, const Tensor *fmha_out,
                                   Tensor *gate_bias_out, int m, int n, int k) {
  auto *gate_weight = ctx.Input<Tensor>("GateWeight");
  auto *gate_bias = ctx.Input<Tensor>("GateBias");

  auto *gate_out = ctx.Output<Tensor>("GateOut");
  gate_out->mutable_data<T>(ctx.GetPlace());

  // The first gate_bias_out stores the result of the multiplication,
  // and the second gate_bias_out stores the result of the multiplication +
  // bias.
  //   gate_bias_out = GEMM(x, gate_weight)
  //   gate_bias_out = gate_bias_out + gate_bias
  auto gate_attn_compute =
      AttnMatMul<T>(ctx.cuda_device_context(), false, false, m, n, k, true);
  gate_attn_compute.ComputeForward(gate_weight, x, gate_bias, gate_bias_out,
                                   gate_bias_out);

  std::vector<const Tensor *> ins = {gate_bias_out, fmha_out};
  std::vector<Tensor *> outs = {gate_out};
  paddle::operators::LaunchSameDimsElementwiseCudaKernel<T>(
      ctx.cuda_device_context(), ins, &outs, SigmoidMultiplyFunctor<T>());
  return gate_out;
}

template <typename T>
Tensor *ComputeGatingLinearBackward(const framework::ExecutionContext &ctx,
                                    const Tensor *fmha_out,
                                    const Tensor *d_gate_out, Tensor *d_x,
                                    Tensor *d_fmha_out, Tensor *d_gate_bias_out,
                                    Tensor *gate_bias_out, int m, int n,
                                    int k) {
  auto *query = ctx.Input<Tensor>("Query");
  auto *gate_weight = ctx.Input<Tensor>("GateWeight");
  auto *gate_bias = ctx.Input<Tensor>("GateBias");

  auto gate_attn_compute =
      AttnMatMul<T>(ctx.cuda_device_context(), false, false, m, n, k, true);
  gate_attn_compute.ComputeForward(gate_weight, query, gate_bias, gate_bias_out,
                                   gate_bias_out);

  auto *d_gate_weight =
      ctx.Output<Tensor>(framework::GradVarName("GateWeight"));
  auto *d_gate_bias = ctx.Output<Tensor>(framework::GradVarName("GateBias"));

  d_gate_weight->mutable_data<T>(ctx.GetPlace());
  d_gate_bias->mutable_data<T>(ctx.GetPlace());

  // Gradient of sigmoid(gate_bias_out) * fmha_out
  std::vector<const Tensor *> ins = {d_gate_out, gate_bias_out, fmha_out};
  std::vector<Tensor *> outs = {d_gate_bias_out, d_fmha_out};
  paddle::operators::LaunchSameDimsElementwiseCudaKernel<
      T, SigmoidMultiplyGradFunctor<T>, 2>(
      ctx.cuda_device_context(), ins, &outs, SigmoidMultiplyGradFunctor<T>());

  gate_attn_compute.ComputeBackward(query, gate_weight, d_gate_bias_out, d_x,
                                    d_gate_weight, d_gate_bias);
  return d_fmha_out;
}

template <typename T>
Tensor *ComputeOutputLinearForward(const framework::ExecutionContext &ctx,
                                   const Tensor *fmha_or_gate_out, int m, int n,
                                   int k) {
  auto *out_linear_weight = ctx.Input<Tensor>("OutLinearWeight");
  auto *out_linear_bias = ctx.Input<Tensor>("OutLinearBias");

  auto *out = ctx.Output<Tensor>("Out");
  out->mutable_data<T>(ctx.GetPlace());

  // out = GEMM(gate_out, out_linear_weight)
  // out = out + out_linear_bias
  auto out_linear_compute =
      AttnMatMul<T>(ctx.cuda_device_context(), false, false, m, n, k, true);
  out_linear_compute.ComputeForward(out_linear_weight, fmha_or_gate_out,
                                    out_linear_bias, out, out);
  return out;
}

template <typename T>
Tensor *ComputeOutputLinearBackward(const framework::ExecutionContext &ctx,
                                    const Tensor *fmha_out, Tensor *d_fmha_out,
                                    int m, int n, int k, bool has_gating) {
  auto *d_out = ctx.Input<Tensor>(framework::GradVarName("Out"));
  auto *out_linear_weight = ctx.Input<Tensor>("OutLinearWeight");
  auto *input = has_gating ? ctx.Input<Tensor>("GateOut") : fmha_out;

  auto *d_out_linear_weight =
      ctx.Output<Tensor>(framework::GradVarName("OutLinearWeight"));
  auto *d_out_linear_bias =
      ctx.Output<Tensor>(framework::GradVarName("OutLinearBias"));
  auto *d_input = has_gating
                      ? ctx.Output<Tensor>(framework::GradVarName("GateOut"))
                      : d_fmha_out;

  d_out_linear_weight->mutable_data<T>(ctx.GetPlace());
  d_out_linear_bias->mutable_data<T>(ctx.GetPlace());
  d_input->mutable_data<T>(ctx.GetPlace());

  auto out_linear_compute =
      AttnMatMul<T>(ctx.cuda_device_context(), false, false, m, n, k, true);
  out_linear_compute.ComputeBackward(input, out_linear_weight, d_out, d_input,
                                     d_out_linear_weight, d_out_linear_bias);
  return d_input;
}

template <typename T>
class FusedGateAttentionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *query = ctx.Input<Tensor>("Query");
    auto *key = ctx.Input<Tensor>("Key");
    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto *nonbatched_bias = ctx.Input<Tensor>("NonbatchedBias");

    auto *q_out = ctx.Output<Tensor>("QueryOut");
    auto *k_out = ctx.Output<Tensor>("KeyOut");
    auto *v_out = ctx.Output<Tensor>("ValueOut");
    auto *qkv_out = ctx.Output<Tensor>("QKVOut");

    const auto merge_qkv = ctx.Attr<bool>("merge_qkv");
    const auto has_gating = ctx.Attr<bool>("has_gating");

    const auto input_q_dims = query->dims();
    int num_head, key_dim, m_size, kv_dim;
    // if self-attention seq_len_r = m_size q_dim = kv_dim
    int batch_size = input_q_dims[0];
    int seq_len_m = input_q_dims[1];
    int seq_len_r = input_q_dims[2];
    int q_dim = input_q_dims[3];
    if (merge_qkv) {
      // x: qkv's input [batch_size, seq_len_m, seq_len_r, qkv_dim]
      // qkv_weight[3, num_head, key_dim, qkv_dim]
      const auto &qkv_w_dims = ctx.Input<Tensor>("QKVWeight")->dims();
      qkv_out->mutable_data<T>(ctx.GetPlace());
      num_head = qkv_w_dims[1];
      key_dim = qkv_w_dims[2];
      m_size = seq_len_r;
      kv_dim = q_dim;
    } else {
      const auto input_k_dims = key->dims();
      q_out->mutable_data<T>(ctx.GetPlace());
      k_out->mutable_data<T>(ctx.GetPlace());
      v_out->mutable_data<T>(ctx.GetPlace());

      const auto q_w_dims = ctx.Input<Tensor>("QueryWeight")->dims();
      num_head = q_w_dims[1];
      key_dim = q_w_dims[2];
      m_size = input_k_dims[2];
      kv_dim = input_k_dims[3];
    }
    auto *softmax_out = ctx.Output<Tensor>("SoftmaxOut");
    auto *qktv_out = ctx.Output<Tensor>("QKTVOut");

    qktv_out->mutable_data<T>(ctx.GetPlace());
    softmax_out->mutable_data<T>(ctx.GetPlace());

    Tensor qkv_transpose_out, q_transpose_out, k_transpose_out, v_transpose_out;
    if (merge_qkv) {
      // 1. Merged QKV Matmul: einsum(nbhqk,nbkhc -> nbqhc)
      //    [batch_size * seq_len_m * seq_len_r * 3 * num_head * c]
      int m = batch_size * seq_len_m * seq_len_r;
      int n = 3 * num_head * key_dim;
      int k = q_dim;
      ComputeMergedQKVMatmulForward<T>(ctx, query, qkv_out, m, n, k);
      qkv_transpose_out.Resize(
          {3, batch_size, seq_len_m, num_head, seq_len_r, q_dim});
      qkv_transpose_out.mutable_data<T>(ctx.GetPlace());
    } else {
      // 1. Separated QKV Matmul
      int q_m = batch_size * seq_len_m * seq_len_r;
      int q_n = num_head * key_dim;
      int q_k = q_dim;

      int k_m = batch_size * seq_len_m * m_size;
      int k_n = num_head * key_dim;
      int k_k = kv_dim;

      ComputeQKVMatmulForward<T>(ctx, query, key, q_out, k_out, v_out, q_m, q_n,
                                 q_k, k_m, k_n, k_k);

      q_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, seq_len_r, key_dim});
      q_transpose_out.mutable_data<T>(ctx.GetPlace());

      k_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, m_size, key_dim});
      k_transpose_out.mutable_data<T>(ctx.GetPlace());

      v_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, m_size, key_dim});
      v_transpose_out.mutable_data<T>(ctx.GetPlace());
    }

    // 2. FMHA
    Tensor fmha_out;
    fmha_out.Resize({batch_size, seq_len_m, seq_len_r, num_head, key_dim});
    fmha_out.mutable_data<T>(ctx.GetPlace());
    auto fmha_compute =
        FMHAGateRef<T>(ctx.cuda_device_context(), merge_qkv, batch_size,
                       seq_len_m, seq_len_r, m_size, num_head, key_dim);

    fmha_compute.ComputeForward(
        nonbatched_bias, *q_out, *k_out, *v_out, *qkv_out, src_mask,
        &q_transpose_out, &k_transpose_out, &v_transpose_out,
        &qkv_transpose_out, softmax_out, qktv_out, &fmha_out);

    // 3. Gating Linear
    Tensor *fmha_or_gate_out = nullptr;
    if (has_gating) {
      int m = batch_size * seq_len_m * seq_len_r;
      int n = num_head * key_dim;
      int k = q_dim;

      Tensor gate_bias_out;
      gate_bias_out.Resize(
          {batch_size, seq_len_m, seq_len_r, num_head, key_dim});
      gate_bias_out.mutable_data<T>(ctx.GetPlace());
      fmha_or_gate_out = ComputeGatingLinearForward<T>(ctx, query, &fmha_out,
                                                       &gate_bias_out, m, n, k);
    } else {
      fmha_or_gate_out = &fmha_out;
    }

    // 4. Output Linear
    int m = batch_size * seq_len_m * seq_len_r;
    int n = q_dim;
    int k = num_head * key_dim;
    ComputeOutputLinearForward<T>(ctx, fmha_or_gate_out, m, n, k);
  }
};

template <typename T>
class FusedGateAttentionGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    const auto has_gating = ctx.Attr<bool>("has_gating");
    const auto merge_qkv = ctx.Attr<bool>("merge_qkv");

    // forward input
    auto *query = ctx.Input<Tensor>("Query");
    auto *key = ctx.Input<Tensor>("Key");
    auto *nonbatched_bias = ctx.Input<Tensor>("NonbatchedBias");

    // forward output, backward input
    auto *softmax_out = ctx.Input<Tensor>("SoftmaxOut");
    auto *qktv_out = ctx.Input<Tensor>("QKTVOut");
    auto *q_out = ctx.Input<Tensor>("QueryOut");
    auto *key_out = ctx.Input<Tensor>("KeyOut");
    auto *value_out = ctx.Input<Tensor>("ValueOut");
    auto *qkv_out = ctx.Input<Tensor>("QKVOut");

    // backward output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("Query"));
    auto *d_key = ctx.Output<Tensor>(framework::GradVarName("Key"));
    auto *d_qktv_out = ctx.Output<Tensor>(framework::GradVarName("QKTVOut"));
    auto *d_softmax_out =
        ctx.Output<Tensor>(framework::GradVarName("SoftmaxOut"));
    auto *d_qkv_out = ctx.Output<Tensor>(framework::GradVarName("QKVOut"));
    auto *d_q_out = ctx.Output<Tensor>(framework::GradVarName("QueryOut"));
    auto *d_k_out = ctx.Output<Tensor>(framework::GradVarName("KeyOut"));
    auto *d_v_out = ctx.Output<Tensor>(framework::GradVarName("ValueOut"));

    d_x->mutable_data<T>(ctx.GetPlace());
    d_qktv_out->mutable_data<T>(ctx.GetPlace());
    d_softmax_out->mutable_data<T>(ctx.GetPlace());

    // parameter grad
    auto *d_nonbatched_bias =
        ctx.Output<Tensor>(framework::GradVarName("NonbatchedBias"));

    if (nonbatched_bias) {
      d_nonbatched_bias->mutable_data<T>(ctx.GetPlace());
    }

    if (key) {
      d_key->mutable_data<T>(ctx.GetPlace());
    }

    const auto &input_q_dims = query->dims();
    int num_head, key_dim, m_size, kv_dim;
    int batch_size = input_q_dims[0];
    int seq_len_m = input_q_dims[1];
    int seq_len_r = input_q_dims[2];
    int q_dim = input_q_dims[3];

    if (merge_qkv) {
      // qkv_weight[3, n_head, c, qkv_dim]
      const auto &qkv_w_dims = ctx.Input<Tensor>("QKVWeight")->dims();
      d_qkv_out->mutable_data<T>(ctx.GetPlace());
      num_head = qkv_w_dims[1];
      key_dim = qkv_w_dims[2];
      m_size = seq_len_r;
      kv_dim = q_dim;
    } else {
      const auto &input_k_dims = key->dims();
      const auto &q_w_dims = ctx.Input<Tensor>("QueryWeight")->dims();
      d_q_out->mutable_data<T>(ctx.GetPlace());
      d_k_out->mutable_data<T>(ctx.GetPlace());
      d_v_out->mutable_data<T>(ctx.GetPlace());
      num_head = q_w_dims[1];
      key_dim = q_w_dims[2];
      m_size = input_k_dims[2];
      kv_dim = input_k_dims[3];
    }

    // Re-compute the fmha_out.
    Tensor fmha_out;
    fmha_out.Resize({batch_size, seq_len_m, seq_len_r, num_head, key_dim});
    fmha_out.mutable_data<T>(ctx.GetPlace());

    auto fmha_compute =
        FMHAGateRef<T>(ctx.cuda_device_context(), merge_qkv, batch_size,
                       seq_len_m, seq_len_r, m_size, num_head, key_dim);
    fmha_compute.ComputeQKTVTransposeForward(*qktv_out, &fmha_out);

    // 1. Gradient of Output Linear
    int m = batch_size * seq_len_m * seq_len_r;
    int n = q_dim;
    int k = num_head * key_dim;

    Tensor d_fmha_out;
    d_fmha_out.Resize({batch_size, seq_len_m, seq_len_r, num_head, key_dim});
    d_fmha_out.mutable_data<T>(ctx.GetPlace());

    Tensor *d_fhma_or_gate_out = ComputeOutputLinearBackward<T>(
        ctx, &fmha_out, &d_fmha_out, m, n, k, has_gating);

    // 2. Gradient of Gating Linear
    if (has_gating) {
      m = batch_size * seq_len_m * seq_len_r;
      n = num_head * key_dim;
      k = q_dim;

      Tensor gate_bias_out;
      gate_bias_out.Resize(
          {batch_size, seq_len_m, seq_len_r, num_head, key_dim});
      gate_bias_out.mutable_data<T>(ctx.GetPlace());

      Tensor d_gate_bias_out;
      d_gate_bias_out.Resize(
          {batch_size, seq_len_m, seq_len_r, num_head, key_dim});
      d_gate_bias_out.mutable_data<T>(ctx.GetPlace());
      // d_fhma_or_gate_out is d_gate_out.
      ComputeGatingLinearBackward<T>(ctx, &fmha_out, d_fhma_or_gate_out, d_x,
                                     &d_fmha_out, &d_gate_bias_out,
                                     &gate_bias_out, m, n, k);
    }

    // Re-compute the qkv_transpose_out.
    Tensor qkv_transpose_out, d_qkv_transpose_out;
    Tensor q_transpose_out, k_transpose_out, v_transpose_out, d_q_transpose_out,
        d_k_transpose_out, d_v_transpose_out;
    if (merge_qkv) {
      qkv_transpose_out.Resize(
          {3, batch_size, seq_len_m, num_head, seq_len_r, key_dim});
      qkv_transpose_out.mutable_data<T>(ctx.GetPlace());
      fmha_compute.ComputeQKVTransposeForward(*qkv_out, &qkv_transpose_out);

      d_qkv_transpose_out.Resize(
          {3, batch_size, seq_len_m, num_head, seq_len_r, key_dim});
      d_qkv_transpose_out.mutable_data<T>(ctx.GetPlace());
    } else {
      q_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, seq_len_r, key_dim});
      q_transpose_out.mutable_data<T>(ctx.GetPlace());

      k_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, m_size, key_dim});
      k_transpose_out.mutable_data<T>(ctx.GetPlace());

      v_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, m_size, key_dim});
      v_transpose_out.mutable_data<T>(ctx.GetPlace());
      fmha_compute.ComputeQKVTransposeForward(
          *q_out, *key_out, *value_out, &q_transpose_out, &k_transpose_out,
          &v_transpose_out);

      d_q_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, seq_len_r, key_dim});
      d_q_transpose_out.mutable_data<T>(ctx.GetPlace());

      d_k_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, m_size, key_dim});
      d_k_transpose_out.mutable_data<T>(ctx.GetPlace());

      d_v_transpose_out.Resize(
          {batch_size, seq_len_m, num_head, m_size, key_dim});
      d_v_transpose_out.mutable_data<T>(ctx.GetPlace());
    }

    // 3. Gradient of FMHA
    fmha_compute.ComputeBackward(
        q_transpose_out, k_transpose_out, v_transpose_out, qkv_transpose_out,
        *softmax_out, d_fmha_out, nonbatched_bias, d_nonbatched_bias,
        d_qktv_out, d_softmax_out, nullptr, &d_q_transpose_out,
        &d_k_transpose_out, &d_v_transpose_out, d_q_out, d_k_out, d_v_out,
        &d_qkv_transpose_out, d_qkv_out);

    bool use_addto = has_gating ? true : false;
    if (merge_qkv) {
      // 4. Gradient of Merged QKV Matmul
      m = batch_size * seq_len_m * seq_len_r;
      n = 3 * num_head * key_dim;
      k = q_dim;
      ComputeMergedQKVMatmulBackward<T>(ctx, query, d_qkv_out, d_x, m, n, k,
                                        use_addto);
      if (key) {
        d_key = d_x;
      }
    } else {
      // 4. Gradient of Separated QKV Matmul
      int q_m = batch_size * seq_len_m * seq_len_r;
      int q_n = num_head * key_dim;
      int q_k = q_dim;

      int k_m = batch_size * seq_len_m * m_size;
      int k_n = num_head * key_dim;
      int k_k = kv_dim;
      ComputeQKVMatmulBackward<T>(ctx, query, key, d_q_out, d_k_out, d_v_out,
                                  d_x, d_key, q_m, q_n, q_k, k_m, k_n, k_k,
                                  use_addto);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(fused_gate_attention,
                        ops::FusedGateAttentionOpKernel<float>,
                        ops::FusedGateAttentionOpKernel<double>,
                        ops::FusedGateAttentionOpKernel<plat::float16>,
                        ops::FusedGateAttentionOpKernel<plat::bfloat16>);
REGISTER_OP_CUDA_KERNEL(fused_gate_attention_grad,
                        ops::FusedGateAttentionGradKernel<float>,
                        ops::FusedGateAttentionGradKernel<double>,
                        ops::FusedGateAttentionGradKernel<plat::float16>,
                        ops::FusedGateAttentionGradKernel<plat::bfloat16>);
