#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <time.h>

#include <random>
#include <vector>

#include "gtest/gtest.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/framework/program_desc.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/fused/fused_residual_dropout_bias.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/string/printf.h"

namespace framework = paddle::framework;
namespace platform = paddle::platform;

USE_OP(elementwise_add);
USE_OP(dropout);

template <typename T>
void Dropout(const std::vector<T> &x, const framework::DDim &x_dim,
             std::vector<T> *out, std::vector<uint8_t> *mask,
             const platform::CUDADeviceContext &ctx, uint64_t seed,
             float dropout_prob, bool is_upscale_in_train, bool is_test) {
  framework::Scope scope;
  auto var_x = scope.Var("X");
  auto tensor_x = var_x->GetMutable<framework::LoDTensor>();
  tensor_x->Resize(x_dim);
  tensor_x->mutable_data<T>(ctx.GetPlace());
  hipMemcpy(tensor_x->data<T>(), x.data(), x_dim[0] * x_dim[1] * sizeof(T),
             hipMemcpyHostToDevice);

  auto var_out = scope.Var("Out");
  auto tensor_out = var_out->GetMutable<framework::LoDTensor>();

  auto var_mask = scope.Var("Mask");
  auto tensor_mask = var_mask->GetMutable<framework::LoDTensor>();

  framework::AttributeMap attrs;
  attrs.insert({"fix_seed", 1});
  attrs.insert({"seed", static_cast<int>(seed)});
  attrs.insert({"dropout_prob", dropout_prob});
  if (is_upscale_in_train) {
    attrs.insert({"dropout_implementation", std::string("upscale_in_train")});
  }
  if (is_test) {
    attrs.insert({"is_test", 1});
  }

  auto op = framework::OpRegistry::CreateOp(
      "dropout", {{"X", {"X"}}}, {{"Out", {"Out"}}, {"Mask", {"Mask"}}}, attrs);
  op->Run(scope, ctx.GetPlace());
  hipMemcpy((*out).data(), tensor_out->data<T>(),
             x_dim[0] * x_dim[1] * sizeof(T), hipMemcpyDeviceToHost);
  if (!is_test) {
    hipMemcpy((*mask).data(), tensor_mask->data<uint8_t>(),
               x_dim[0] * x_dim[1] * sizeof(uint8_t), hipMemcpyDeviceToHost);
  }
  ctx.Wait();
}

template <typename T>
void DropoutGrad(std::vector<T> *dx, const framework::DDim &x_dim,
                 const std::vector<T> &dout, const std::vector<uint8_t> &mask,
                 const platform::CUDADeviceContext &ctx, float dropout_prob,
                 bool is_upscale_in_train) {
  framework::Scope scope;
  const size_t n = x_dim[0] * x_dim[1];
  auto var_out = scope.Var("DOut");
  auto tensor_out = var_out->GetMutable<framework::LoDTensor>();
  tensor_out->Resize(x_dim);
  tensor_out->mutable_data<T>(ctx.GetPlace());
  hipMemcpy(tensor_out->data<T>(), dout.data(), n * sizeof(T),
             hipMemcpyHostToDevice);

  auto var_mask = scope.Var("Mask");
  auto tensor_mask = var_mask->GetMutable<framework::LoDTensor>();
  tensor_mask->Resize(x_dim);
  tensor_mask->mutable_data<uint8_t>(ctx.GetPlace());
  hipMemcpy(tensor_mask->data<uint8_t>(), mask.data(), n * sizeof(uint8_t),
             hipMemcpyHostToDevice);

  auto var_dx = scope.Var("DX");
  auto tensor_dx = var_dx->GetMutable<framework::LoDTensor>();

  framework::AttributeMap attrs;
  attrs.insert({"dropout_prob", dropout_prob});
  attrs.insert({"is_test", 0});
  if (is_upscale_in_train) {
    attrs.insert({"dropout_implementation", std::string("upscale_in_train")});
  } else {
    attrs.insert({"dropout_implementation", std::string("downgrade_in_infer")});
  }

  auto op = framework::OpRegistry::CreateOp(
      "dropout_grad", {{"Out@GRAD", {"DOut"}}, {"Mask", {"Mask"}}},
      {{"X@GRAD", {"DX"}}}, attrs);
  op->Run(scope, ctx.GetPlace());

  hipMemcpy((*dx).data(), tensor_dx->data<T>(),
             x_dim[0] * x_dim[1] * sizeof(T), hipMemcpyDeviceToHost);
  ctx.Wait();
}

template <typename T>
struct TestFusedResidualDropoutBias {
  uint32_t _rows;
  uint32_t _cols;
  uint64_t _seed;
  float _dropout_prob;
  bool _is_upscale_in_train;
  bool _is_test;  // default false,  Set to true for inference only
  bool _has_bias = true;
  framework::Tensor _src, _residual, _bias, _out, _mask;
  framework::Tensor _dsrc, _dbias;

  std::vector<T> _src_vec, _residual_vec, _bias_vec, _out_vec, _mask_vec;
  std::vector<T> _correct_out, _correct_dsrc, _correct_dbias;
  std::vector<uint8_t> _correct_mask;

  platform::CUDAPlace _place;
  platform::CUDADeviceContext *_ctx;

  TestFusedResidualDropoutBias() {
    _rows = 32;
    _cols = 32;
    _seed = 0;
    _dropout_prob = 0.0;
    _is_upscale_in_train = false;
    _is_test = false;
    _has_bias = true;
    _ctx = new platform::CUDADeviceContext(_place);
  }

  TestFusedResidualDropoutBias(int rows, int cols, uint64_t seed = 0,
                               float dropout_prob = 0.0,
                               bool is_upscale_in_train = false,
                               bool is_test = false) {
    _rows = rows;
    _cols = cols;
    _seed = seed;
    _dropout_prob = dropout_prob;
    _is_upscale_in_train = is_upscale_in_train;
    _is_test = is_test;
    _has_bias = true;
    _ctx = new platform::CUDADeviceContext(_place);
  }

  ~TestFusedResidualDropoutBias() { delete _ctx; }

  void SetUp() {
    const int n = _rows * _cols;
    _correct_out.resize(n);
    _correct_mask.resize(n);
    _correct_dsrc.resize(n);
    _correct_dbias.resize(_cols);

    _src_vec.resize(n);
    _residual_vec.resize(n);
    _bias_vec.resize(_cols);
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> dis(0.0, 1.0);

    for (int i = 0; i < _rows; i++) {
      for (int j = 0; j < _cols; j++) {
        _src_vec[i * _cols + j] = static_cast<T>(dis(random));
        _residual_vec[i * _cols + j] = static_cast<T>(dis(random));
        if (i == 0) _bias_vec[j] = dis(random);
      }
    }

    framework::TensorFromVector<T>(_src_vec, *_ctx, &_src);
    _src.Resize({_rows, _cols});
    framework::TensorFromVector<T>(_residual_vec, *_ctx, &_residual);
    _residual.Resize({_rows, _cols});
    if (_has_bias) {
      framework::TensorFromVector<T>(_bias_vec, *_ctx, &_bias);
      _bias.Resize({_cols});
    }

    {
      _out.Resize({_rows, _cols});
      _out.mutable_data<T>(_place);
      _mask.Resize({_rows, _cols});
      _mask.mutable_data<uint8_t>(_place);
      _dsrc.Resize({_rows, _cols});
      _dsrc.mutable_data<T>(_place);

      if (_has_bias) {
        _dbias.Resize({_cols});
        _dbias.mutable_data<T>(_place);
      }
    }
  }

  void BaseForward() {
    std::vector<T> out1(_rows * _cols), out2(_rows * _cols);
    if (_has_bias) {
      for (int i = 0; i < _rows; i++) {
        for (int j = 0; j < _cols; j++) {
          out1[i * _cols + j] = _src_vec[i * _cols + j] + _bias_vec[j];
        }
      }
      Dropout<T>(out1, _src.dims(), &out2, &_correct_mask, *_ctx, _seed,
                 _dropout_prob, _is_upscale_in_train, _is_test);
    } else {
      Dropout<T>(_src_vec, _src.dims(), &out2, &_correct_mask, *_ctx, _seed,
                 _dropout_prob, _is_upscale_in_train, _is_test);
    }
    for (int i = 0; i < _rows; i++) {
      for (int j = 0; j < _cols; j++) {
        _correct_out[i * _cols + j] =
            _residual_vec[i * _cols + j] + out2[i * _cols + j];
      }
    }
    _ctx->Wait();
  }

  void BaseBackward() {
    if (!_is_upscale_in_train) {
      for (int i = 0; i < _rows * _cols; i++) {
        _correct_dsrc[i] = _correct_out[i] * static_cast<T>(_correct_mask[i]);
      }
    } else {
      DropoutGrad<T>(&_correct_dsrc, _src.dims(), _correct_out, _correct_mask,
                     *_ctx, _dropout_prob, _is_upscale_in_train);
    }
    memset(&_correct_dbias[0], 0, _cols * sizeof(T));
    for (int i = 0; i < _rows; i++) {
      for (int j = 0; j < _cols; j++) {
        _correct_dbias[j] += _correct_out[i * _cols + j];
      }
    }
  }

  void FusedForward() {
    auto threads = paddle::operators::GetResidualDropoutBiasThreads(
        *_ctx, (uint64_t)_rows, (uint64_t)_cols);
    const int increment =
        ((_cols - 1) / (threads.first.x * threads.second.x * VecSize) + 1) *
        VecSize;

    T *bias_ptr = nullptr;
    if (_has_bias) {
      bias_ptr = _bias.data<T>();
    }
    if (_is_test) {
      paddle::operators::LaunchResidualDropoutBiasTest<T>(
          _rows, _cols, _dropout_prob, _is_upscale_in_train, _src.data<T>(),
          _residual.data<T>(), bias_ptr, _out.data<T>(), *_ctx);
    } else {
      paddle::operators::LaunchResidualDropoutBias<T, uint8_t>(
          _rows, _cols, increment, _seed, _dropout_prob, _is_upscale_in_train,
          _src.data<T>(), _residual.data<T>(), bias_ptr, _mask.data<uint8_t>(),
          _out.data<T>(), *_ctx);
    }
    _ctx->Wait();
  }

  void FusedBackward() {
    if (_is_test) return;

    T *bias_ptr = nullptr;
    if (_has_bias) {
      bias_ptr = _dbias.data<T>();
    }
    paddle::operators::LaunchResidualDropoutBiasGrad<T, uint8_t>(
        _out.data<T>(), _mask.data<uint8_t>(), _dropout_prob,
        _is_upscale_in_train, _rows, _cols, _dsrc.data<T>(), bias_ptr, *_ctx);
  }

  void Run() {
    SetUp();
    BaseForward();
    FusedForward();
    BaseBackward();
    FusedBackward();
  }

  void CheckOut(const T diff) {
    const int n = _rows * _cols;
    std::vector<T> out(n);
    std::vector<uint8_t> mask(n);
    hipMemcpy(out.data(), _out.data<T>(), _rows * _cols * sizeof(T),
               hipMemcpyDeviceToHost);
    if (!_is_test) {
      hipMemcpy(mask.data(), _mask.data<uint8_t>(),
                 _rows * _cols * sizeof(uint8_t), hipMemcpyDeviceToHost);
    }
    _ctx->Wait();

    for (int i = 0; i < n; i++) {
      EXPECT_LT(std::abs(out[i] - _correct_out[i]), diff);
      if (!_is_test) EXPECT_EQ(mask[i], _correct_mask[i]);
    }
  }

  void CheckGrad(const T diff) {
    if (_is_test) return;

    const int n = _rows * _cols;

    std::vector<T> dsrc(n);
    hipMemcpy(dsrc.data(), _dsrc.data<T>(), _rows * _cols * sizeof(T),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
      EXPECT_LT(std::abs(dsrc[i] - _correct_dsrc[i]), diff);
    }

    if (_has_bias) {
      std::vector<T> dbias(_cols);
      hipMemcpy(dbias.data(), _dbias.data<T>(), _cols * sizeof(T),
                 hipMemcpyDeviceToHost);
      _ctx->Wait();
      for (int i = 0; i < _cols; i++) {
        EXPECT_LT(std::abs(dbias[i] - _correct_dbias[i]), diff);
      }
    }
  }
};

TEST(FusedDropout, GPUFusedRedisualDorpoutBias) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBiasDouble) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<double> test(rows, cols);
  test.Run();
  test.CheckOut(static_cast<double>(1e-5));
  test.CheckGrad(static_cast<double>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBiasFp16) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<platform::float16> test(rows, cols);
  test.Run();
  test.CheckOut(static_cast<platform::float16>(1e-2));
  // For inference, check_grad is not required. ref: test_dropout_op.py
  // test.CheckGrad((platform::float16)1e-2);
}

// test no bias and cols % 4 == 0
TEST(FusedDropout, GPUFusedRedisualDorpoutBiasNoBias) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols);
  test._has_bias = false;
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

// test no bias and cols % 4 != 0
TEST(FusedDropout, GPUFusedRedisualDorpoutBiasNoBias2) {
  const int rows = 16;
  const int cols = 17;
  TestFusedResidualDropoutBias<float> test(rows, cols);
  test._has_bias = false;
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

// test add bias and cols % 4 != 0
TEST(FusedDropout, GPUFusedRedisualDorpoutBias2) {
  const int rows = 16;
  const int cols = 17;
  TestFusedResidualDropoutBias<float> test(rows, cols);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBias3) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 0, 1.0, false, false);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBias4) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 0, 1.0, false, false);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBias5) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 0, 1.0, true, false);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBias6) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 0, 0.35, true, true);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}

TEST(FusedDropout, GPUFusedRedisualDorpoutBias7) {
  const int rows = 16;
  const int cols = 16;
  TestFusedResidualDropoutBias<float> test(rows, cols, 125, 0.0, false, false);
  test.Run();
  test.CheckOut(static_cast<float>(1e-5));
  test.CheckGrad(static_cast<float>(1e-5));
}
