#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/fused/cudnn_bn_stats_finalize.cu.h"
#include "paddle/fluid/operators/fused/cudnn_norm_conv.cu.h"
#include "paddle/fluid/operators/fused/cudnn_scale_bias_add_relu.cu.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
class ResNetUnitKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    PADDLE_ENFORCE_EQ(platform::CudnnDataType<T>::type, HIPDNN_DATA_HALF,
                      platform::errors::Unavailable(
                          "ResNetUnitOp only supports float16 for now."));

    // input x
    const Tensor *input_x = ctx.Input<Tensor>("X");
    const Tensor *filter_x = ctx.Input<Tensor>("FilterX");
    const Tensor *scale_x = ctx.Input<Tensor>("ScaleX");
    const Tensor *bias_x = ctx.Input<Tensor>("BiasX");
    // norm conv
    Tensor *conv_out_x = ctx.Output<Tensor>("ConvX");
    // bn finalize
    Tensor *saved_mean_x = ctx.Output<Tensor>("SavedMeanX");
    Tensor *saved_invstd_x = ctx.Output<Tensor>("SavedInvstdX");
    Tensor *running_mean_x = ctx.Output<Tensor>("RunningMeanX");
    Tensor *running_var_x = ctx.Output<Tensor>("RunningVarX");
    // sbar
    Tensor *output = ctx.Output<Tensor>("Y");
    Tensor *bitmask = ctx.Output<Tensor>("BitMask");
    // attrs
    int padding = ctx.Attr<int>("padding");
    int stride = ctx.Attr<int>("stride");
    int stride_z = ctx.Attr<int>("stride_z");
    int dilation = ctx.Attr<int>("dilation");
    int group = ctx.Attr<int>("group");
    double eps = static_cast<double>(ctx.Attr<float>("epsilon"));
    double momentum = static_cast<double>(ctx.Attr<float>("momentum"));
    bool has_shortcut = ctx.Attr<bool>("has_shortcut");
    bool fuse_add = ctx.Attr<bool>("fuse_add");
    bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    bool is_test = ctx.Attr<bool>("is_test");
    bool is_train = !is_test && !use_global_stats;
    std::string act_type = ctx.Attr<std::string>("act_type");

    auto input_x_shape = phi::vectorize<int>(input_x->dims());
    auto filter_x_shape = phi::vectorize<int>(filter_x->dims());
    // std::swap used to convert shape of filter from conv2d when kernel size is
    // 1.
    if (filter_x_shape[1] != filter_x_shape[2] && 1 == filter_x_shape[2]) {
      std::swap(filter_x_shape[1], filter_x_shape[3]);
    }
    auto param_dims = scale_x->dims();
    auto param_shape = phi::vectorize<int>(scale_x->dims());
    if (1 == param_shape.size()) {
      param_shape = {1, 1, 1, param_shape[0]};
    }
    auto output_shape = phi::vectorize<int>(output->dims());
    auto bitmask_shape = phi::vectorize<int>(bitmask->dims());
    int output_channel = filter_x_shape[0];
    int64_t ele_count =
        std::accumulate(output_shape.begin(), output_shape.end(), 1,
                        std::multiplies<int>()) /
        output_channel;

    auto place = ctx.GetPlace();
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    // 1. Conv
    Tensor sum_x;
    Tensor sum_of_squares_x;
    sum_x.Resize(param_dims);
    sum_of_squares_x.Resize(param_dims);
    CudnnNormConvolution<T> conv_x_op(dev_ctx, input_x_shape, filter_x_shape,
                                      output_shape, padding, stride, dilation,
                                      group);
    conv_x_op.Forward(dev_ctx, *input_x, *filter_x, conv_out_x, &sum_x,
                      &sum_of_squares_x);

    // 2. BN
    Tensor equiv_scale_x;
    Tensor equiv_bias_x;
    equiv_scale_x.Resize(param_dims);
    equiv_bias_x.Resize(param_dims);
    CudnnBNStatsFinalize<T> bn_x_op(dev_ctx, param_shape);
    bn_x_op.Forward(dev_ctx, sum_x, sum_of_squares_x, *scale_x, *bias_x,
                    saved_mean_x, saved_invstd_x, running_mean_x, running_var_x,
                    &equiv_scale_x, &equiv_bias_x, eps, momentum, ele_count,
                    is_train);

    // 3. scale + bias + add + relu
    CudnnScaleBiasAddRelu<T> sbar_op(dev_ctx, act_type, fuse_add, has_shortcut,
                                     output_shape, param_shape, bitmask_shape);
    if (has_shortcut) {
      // input z
      const Tensor *input_z = ctx.Input<Tensor>("Z");
      const Tensor *filter_z = ctx.Input<Tensor>("FilterZ");
      const Tensor *scale_z = ctx.Input<Tensor>("ScaleZ");
      const Tensor *bias_z = ctx.Input<Tensor>("BiasZ");
      // norm conv
      Tensor *conv_out_z = ctx.Output<Tensor>("ConvZ");
      // bn finalize
      Tensor *saved_mean_z = ctx.Output<Tensor>("SavedMeanZ");
      Tensor *saved_invstd_z = ctx.Output<Tensor>("SavedInvstdZ");
      Tensor *running_mean_z = ctx.Output<Tensor>("RunningMeanZ");
      Tensor *running_var_z = ctx.Output<Tensor>("RunningVarZ");

      auto input_z_shape = phi::vectorize<int>(input_z->dims());
      auto filter_z_shape = phi::vectorize<int>(filter_z->dims());

      // 3.1 Conv for second input
      Tensor sum_z;
      Tensor sum_of_squares_z;
      sum_z.Resize(param_dims);
      sum_of_squares_z.Resize(param_dims);
      CudnnNormConvolution<T> conv_z_op(dev_ctx, input_z_shape, filter_z_shape,
                                        output_shape, padding, stride_z,
                                        dilation, group);
      conv_z_op.Forward(dev_ctx, *input_z, *filter_z, conv_out_z, &sum_z,
                        &sum_of_squares_z);

      // 3.2 BN for second input
      Tensor equiv_scale_z;
      Tensor equiv_bias_z;
      equiv_scale_z.Resize(param_dims);
      equiv_bias_z.Resize(param_dims);
      CudnnBNStatsFinalize<T> bn_z_op(dev_ctx, param_shape);
      bn_z_op.Forward(dev_ctx, sum_z, sum_of_squares_z, *scale_z, *bias_z,
                      saved_mean_z, saved_invstd_z, running_mean_z,
                      running_var_z, &equiv_scale_z, &equiv_bias_z, eps,
                      momentum, ele_count, is_train);
      // 3.3 sbar
      sbar_op.Forward(dev_ctx, *conv_out_x, equiv_scale_x, equiv_bias_x,
                      conv_out_z, &equiv_scale_z, &equiv_bias_z, output,
                      bitmask);
    } else {
      const Tensor *input_z = fuse_add ? ctx.Input<Tensor>("Z") : nullptr;
      sbar_op.Forward(dev_ctx, *conv_out_x, equiv_scale_x, equiv_bias_x,
                      input_z, nullptr, nullptr, output, bitmask);
    }
  }
};

template <typename T>
class ResNetUnitGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    PADDLE_ENFORCE_EQ(platform::CudnnDataType<T>::type, HIPDNN_DATA_HALF,
                      platform::errors::Unavailable(
                          "ResNetUnitOp only supports float16 for now."));

    const Tensor *y_grad = ctx.Input<Tensor>(framework::GradVarName("Y"));

    const Tensor *x = ctx.Input<Tensor>("X");
    const Tensor *filter_x = ctx.Input<Tensor>("FilterX");
    const Tensor *scale_x = ctx.Input<Tensor>("ScaleX");
    const Tensor *bias_x = ctx.Input<Tensor>("BiasX");
    const Tensor *saved_mean_x = ctx.Input<Tensor>("SavedMeanX");
    const Tensor *saved_invstd_x = ctx.Input<Tensor>("SavedInvstdX");

    const Tensor *conv_out_x = ctx.Input<Tensor>("ConvX");
    const Tensor *output = ctx.Input<Tensor>("Y");
    const Tensor *bitmask = ctx.Input<Tensor>("BitMask");

    Tensor *x_grad = ctx.Output<Tensor>(framework::GradVarName("X"));
    Tensor *filter_x_grad =
        ctx.Output<Tensor>(framework::GradVarName("FilterX"));
    Tensor *scale_x_grad = ctx.Output<Tensor>(framework::GradVarName("ScaleX"));
    Tensor *bias_x_grad = ctx.Output<Tensor>(framework::GradVarName("BiasX"));

    int padding = ctx.Attr<int>("padding");
    int stride = ctx.Attr<int>("stride");
    int stride_z = ctx.Attr<int>("stride_z");
    int dilation = ctx.Attr<int>("dilation");
    int group = ctx.Attr<int>("group");
    double eps = static_cast<double>(ctx.Attr<float>("epsilon"));
    double momentum = static_cast<double>(ctx.Attr<float>("momentum"));
    bool has_shortcut = ctx.Attr<bool>("has_shortcut");
    bool fuse_add = ctx.Attr<bool>("fuse_add");
    bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    std::string act_type = ctx.Attr<std::string>("act_type");

    auto x_shape = phi::vectorize<int>(x->dims());
    auto filter_x_shape = phi::vectorize<int>(filter_x->dims());
    auto param_shape = phi::vectorize<int>(scale_x->dims());
    auto output_shape = phi::vectorize<int>(output->dims());
    auto bitmask_shape = phi::vectorize<int>(bitmask->dims());

    auto place = ctx.GetPlace();
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    // 1. Backward of BN (+ Add + Relu) for x, get conv_out_x_grad,
    // scale_x_grad, bias_x_grad
    Tensor conv_out_x_grad;
    conv_out_x_grad.Resize(conv_out_x->dims());
    CudnnScaleBiasAddRelu<T> sbar_x_op(dev_ctx, act_type, fuse_add,
                                       has_shortcut, output_shape, param_shape,
                                       bitmask_shape);
    if (has_shortcut) {
      //       X                   Z
      //       |                   |
      //    NormConv            NormConv
      //       |                   |
      // BNStatsFinalize    BNStatsFinalize
      //       \                   /
      //          ScaleBiasAddRelu
      //                  |
      //                  Y
      const Tensor *z = ctx.Input<Tensor>("Z");
      const Tensor *filter_z = ctx.Input<Tensor>("FilterZ");
      const Tensor *scale_z = ctx.Input<Tensor>("ScaleZ");
      const Tensor *bias_z = ctx.Input<Tensor>("BiasZ");
      const Tensor *saved_mean_z = ctx.Input<Tensor>("SavedMeanZ");
      const Tensor *saved_invstd_z = ctx.Input<Tensor>("SavedInvstdZ");
      const Tensor *conv_out_z = ctx.Input<Tensor>("ConvZ");

      Tensor *z_grad = ctx.Output<Tensor>(framework::GradVarName("Z"));
      Tensor *filter_z_grad =
          ctx.Output<Tensor>(framework::GradVarName("FilterZ"));
      Tensor *scale_z_grad =
          ctx.Output<Tensor>(framework::GradVarName("ScaleZ"));
      Tensor *bias_z_grad = ctx.Output<Tensor>(framework::GradVarName("BiasZ"));

      // 1.1 Backward of BN + Add (+ Relu) for x, get conv_out_x_grad,
      // scale_x_grad, bias_x_grad and z_grad_temp
      Tensor z_grad_temp;
      z_grad_temp.Resize(conv_out_z->dims());
      sbar_x_op.Backward(dev_ctx, *y_grad, *conv_out_x, *scale_x, *bias_x,
                         *saved_mean_x, *saved_invstd_x, bitmask,
                         &conv_out_x_grad, &z_grad_temp, scale_x_grad,
                         bias_x_grad, eps);

      // 1.2 bn backward for z, get conv_out_z_grad, dscale_z, dbias_z
      Tensor conv_out_z_grad;
      conv_out_z_grad.Resize(conv_out_z->dims());
      CudnnScaleBiasAddRelu<T> sbar_z_op(
          dev_ctx, "", false, false, output_shape, param_shape, bitmask_shape);
      sbar_z_op.Backward(dev_ctx, z_grad_temp, *conv_out_z, *scale_z, *bias_z,
                         *saved_mean_z, *saved_invstd_z, nullptr,
                         &conv_out_z_grad, nullptr, scale_z_grad, bias_z_grad,
                         eps);

      // 1.3 Backward of Conv for z, get z_grad and filter_z_grad
      auto z_shape = phi::vectorize<int>(z->dims());
      auto filter_z_shape = phi::vectorize<int>(filter_z->dims());
      CudnnNormConvolutionGrad<T> conv_z_op(dev_ctx, z_shape, filter_z_shape,
                                            output_shape, padding, stride_z,
                                            dilation, group);
      conv_z_op.Backward(dev_ctx, *z, *filter_z, conv_out_z_grad, z_grad,
                         filter_z_grad);
    } else {
      // 1.1 Backward of BN (+ Add + Relu) for x, get conv_out_x_grad,
      // scale_x_grad, bias_x_grad (and z_grad)
      Tensor *z_grad =
          fuse_add ? ctx.Output<Tensor>(framework::GradVarName("Z")) : nullptr;
      sbar_x_op.Backward(dev_ctx, *y_grad, *conv_out_x, *scale_x, *bias_x,
                         *saved_mean_x, *saved_invstd_x, bitmask,
                         &conv_out_x_grad, z_grad, scale_x_grad, bias_x_grad,
                         eps);
    }

    // 2. Backward of Conv for x, get x_grad and filter_x_grad
    bool use_addto = ctx.Attr<bool>("use_addto");
    CudnnNormConvolutionGrad<T> conv_x_op(dev_ctx, x_shape, filter_x_shape,
                                          output_shape, padding, stride,
                                          dilation, group);
    conv_x_op.Backward(dev_ctx, *x, *filter_x, conv_out_x_grad, x_grad,
                       filter_x_grad, use_addto);
  }
};

}  // namespace operators
}  // namespace paddle

#if CUDNN_VERSION >= 8000
namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(resnet_unit, ops::ResNetUnitKernel<plat::float16>);
REGISTER_OP_CUDA_KERNEL(resnet_unit_grad,
                        ops::ResNetUnitGradKernel<plat::float16>);
#endif
