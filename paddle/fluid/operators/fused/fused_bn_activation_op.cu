// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>

#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/operators/fused/fused_bn_activation_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/norm_utils.h"

PHI_DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace paddle {
namespace operators {
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
class FusedBatchNormActKernel<T, phi::GPUContext>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {}
};

template <typename T>
class FusedBatchNormActGradKernel<T, phi::GPUContext>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {}
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

PD_REGISTER_STRUCT_KERNEL(fused_batch_norm_act_grad,
                          GPU,
                          ALL_LAYOUT,
                          ops::FusedBatchNormActGradKernel,
                          float,
                          double,
                          plat::float16) {}
