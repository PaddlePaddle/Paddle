// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <paddle/fluid/platform/device_context.h>
#include <algorithm>
#include "paddle/fluid/framework/framework.pb.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"
#include "paddle/fluid/operators/math/blas.h"

namespace paddle {
namespace operators {

template <typename DeviceContext, typename T>
class EmbeddingEltWiseLayerNormKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    using Tensor = framework::Tensor;
    auto &device_ctx = context.template device_context<DeviceContext>();
    auto ids = context.MultiInput<framework::Tensor>("Ids");
    auto embs = context.MultiInput<framework::Tensor>("Embs");
    int input_num = static_cast<int>(ids.size());

    framework::Tensor in_ids_(framework::proto::VarType::INT64),
        in_embs_(framework::proto::VarType::INT64);
    framework::DDim in_dim{input_num};
    int device_id;
#ifdef PADDLE_WITH_HIP
    hipGetDevice(&device_id);
#else
    hipGetDevice(&device_id);
#endif
    in_ids_.Resize(in_dim);
    in_embs_.Resize(in_dim);
    int64_t *in_ids_d =
        in_ids_.mutable_data<int64_t>(platform::CUDAPlace(device_id));
    int64_t *in_embs_d =
        in_embs_.mutable_data<int64_t>(platform::CUDAPlace(device_id));

    std::vector<int64_t> in1s, in2s;
    for (int i = 0; i < input_num; ++i) {
      in1s.push_back(reinterpret_cast<uintptr_t>(ids[i]->data<int64_t>()));
      in2s.push_back(reinterpret_cast<uintptr_t>(embs[i]->data<T>()));
    }
#ifdef PADDLE_WITH_HIP
    hipMemcpyAsync(in_ids_d, in1s.data(), sizeof(int64_t) * input_num,
                   hipMemcpyHostToDevice, device_ctx.stream());
    hipMemcpyAsync(in_embs_d, in2s.data(), sizeof(int64_t) * input_num,
                   hipMemcpyHostToDevice, device_ctx.stream());
#else
    hipMemcpyAsync(in_ids_d, in1s.data(), sizeof(int64_t) * input_num,
                    hipMemcpyHostToDevice, device_ctx.stream());
    hipMemcpyAsync(in_embs_d, in2s.data(), sizeof(int64_t) * input_num,
                    hipMemcpyHostToDevice, device_ctx.stream());
#endif

    auto *bias = context.Input<framework::Tensor>("Bias");
    auto *scale = context.Input<framework::Tensor>("Scale");
    auto *out = context.Output<framework::Tensor>("Out");

    // should be (B * S * hidden)
    auto id0_dims = ids[0]->dims();
    auto emb0_dims = embs[0]->dims();

    int batch = id0_dims[0];
    int seq_len = id0_dims[1];
    int hidden = emb0_dims[1];

    auto *bias_d = bias->data<T>();
    auto *scale_d = scale->data<T>();
    auto *output_d = out->mutable_data<T>(context.GetPlace());
    float eps = context.Attr<float>("epsilon");

    int shared_bytes = input_num * sizeof(int64_t);
    math::EmbEltwiseLayerNormFunctor<T> emb_eltwise_layernorm_func;
    emb_eltwise_layernorm_func(batch, seq_len, hidden, in_ids_d, scale_d,
                               bias_d, in_embs_d, output_d, eps, input_num,
                               device_ctx.stream());
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(fused_embedding_eltwise_layernorm,
                        ops::EmbeddingEltWiseLayerNormKernel<
                            paddle::platform::CUDADeviceContext, float>);
