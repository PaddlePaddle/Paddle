/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fused/fused_elemwise_activation_op.h"

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fused_elemwise_activation,
    ops::FusedElemwiseActivationKernel<paddle::platform::CUDADeviceContext,
                                       float>,
    ops::FusedElemwiseActivationKernel<paddle::platform::CUDADeviceContext,
                                       double>,
    ops::FusedElemwiseActivationKernel<paddle::platform::CUDADeviceContext,
                                       paddle::platform::float16>);

REGISTER_OP_CUDA_KERNEL(
    fused_elemwise_activation_grad,
    ops::FusedElemwiseActivationGradKernel<paddle::platform::CUDADeviceContext,
                                           float>,
    ops::FusedElemwiseActivationGradKernel<paddle::platform::CUDADeviceContext,
                                           double>,
    ops::FusedElemwiseActivationGradKernel<paddle::platform::CUDADeviceContext,
                                           paddle::platform::float16>);

REGISTER_OP_CUDA_KERNEL(
    fused_elemwise_add_activation,
    ops::FusedElemwiseActivationKernel<paddle::platform::CUDADeviceContext,
                                       float>,
    ops::FusedElemwiseActivationKernel<paddle::platform::CUDADeviceContext,
                                       double>,
    ops::FusedElemwiseActivationKernel<paddle::platform::CUDADeviceContext,
                                       paddle::platform::float16>);

REGISTER_OP_CUDA_KERNEL(
    fused_elemwise_add_activation_grad,
    ops::FusedElemwiseActivationGradKernel<paddle::platform::CUDADeviceContext,
                                           float>,
    ops::FusedElemwiseActivationGradKernel<paddle::platform::CUDADeviceContext,
                                           double>,
    ops::FusedElemwiseActivationGradKernel<paddle::platform::CUDADeviceContext,
                                           paddle::platform::float16>);
