#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/custom_all_reduce.h"
#include "paddle/fluid/operators/fused/fused_multi_transformer_helper.cu.h"
#include "paddle/fluid/platform/device/gpu/gpu_resource_pool.h"
#include "paddle/phi/kernels/flash_attn_kernel.h"
#include "paddle/phi/kernels/reduce_sum_kernel.h"

#include <chrono>

// #define _DEBUG_FUSED_MULTI_TRANSFORMER
// #define _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR

namespace paddle {
namespace operators {

static phi::DenseTensor CustomAllReduce(const phi::DenseTensor &t) {
  auto *ctx = static_cast<phi::GPUContext *>(
      platform::DeviceContextPool::Instance().Get(t.place()));
  auto comm = GetCustomNCCLComm(*ctx, 0);
  PADDLE_ENFORCE_NOT_NULL(comm);
  phi::DenseTensor ret;
  ret.Resize(t.dims());
  ctx->Alloc(&ret, t.dtype());
  comm->SwapInput(&ret);
  phi::Copy(*ctx, t, t.place(), false, &ret);
  return comm->AllReduce();
}

template <typename T, typename DeviceContext>
class FusedMultiTransformerOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto &dev_ctx = ctx.cuda_device_context();

    auto *time_step = ctx.Input<phi::DenseTensor>("TimeStep");
    // 0. input
    auto *input_x = ctx.Input<phi::DenseTensor>("X");
    const auto input_x_dims = input_x->dims();
    int bsz = input_x_dims[0];
    int seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];
    int bsz_seq = bsz * seq_len;
    const std::string act_method = ctx.Attr<std::string>("act_method");
    bool use_glu = (act_method == "geglu" || act_method == "swiglu");
    const std::string norm_type = ctx.Attr<std::string>("norm_type");
    const bool use_neox_rotary_style = ctx.Attr<bool>("use_neox_rotary_style");
    bool remove_padding = false;
    auto *sequence_lengths = ctx.Input<phi::DenseTensor>("SeqLengths");
    if (sequence_lengths) {
      remove_padding = true;
    }

    auto *beam_cache_offset = ctx.Input<phi::DenseTensor>("BeamCacheOffset");
    int beam_size = 1;
    if (beam_cache_offset) {
      beam_size = beam_cache_offset->dims()[1];
    }

    phi::DenseTensor d_token_tensor;
    phi::DenseTensor padding_offset_tensor;
    phi::DenseTensor x_remove_padding;

    // cumulative seqlens [batch_size+1]
    phi::DenseTensor cu_seqlens_q, cu_seqlens_k;
    bool encoder_remove_padding = (remove_padding && !time_step);
    int token_num = 0;

    auto *out = ctx.Output<phi::DenseTensor>("Out");
    auto *from_data = dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));

    // Init out
    if (encoder_remove_padding) {
      InitValue(dev_ctx, from_data, out->numel(), static_cast<T>(0.));
    }

    // remove padding in encoder
    if (encoder_remove_padding) {
      // just for encoder
      d_token_tensor.Resize({{1}});
      auto *d_token_num = dev_ctx.Alloc<int>(
          &d_token_tensor, d_token_tensor.numel() * sizeof(int));
      // alloc the max size of padding_offset_tensor
      padding_offset_tensor.Resize({{bsz_seq}});
      dev_ctx.Alloc<int>(&padding_offset_tensor,
                         padding_offset_tensor.numel() * sizeof(int));
      cu_seqlens_q.Resize({{bsz + 1}});
      dev_ctx.Alloc<int32_t>(&cu_seqlens_q,
                             cu_seqlens_q.numel() * sizeof(int32_t));

      InvokeGetPaddingOffset(dev_ctx,
                             &token_num,
                             d_token_num,
                             padding_offset_tensor.data<int>(),
                             cu_seqlens_q.data<int>(),
                             sequence_lengths->data<int>(),
                             bsz,
                             seq_len);
      if (token_num == 0) return;
      padding_offset_tensor.Resize({{token_num}});
      x_remove_padding.Resize({{token_num, dim_embed}});
      dev_ctx.Alloc<T>(&x_remove_padding, x_remove_padding.numel() * sizeof(T));
      InvokeRemovePadding(dev_ctx,
                          x_remove_padding.data<T>(),
                          input_x->data<T>(),
                          padding_offset_tensor.data<int>(),
                          token_num,
                          dim_embed);
    } else {
      token_num = bsz_seq;
      if (token_num == 0) return;
    }

    auto *padding_offset_data =
        encoder_remove_padding ? padding_offset_tensor.data<int>() : nullptr;

    // 1. layer norm
    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    const float epsilon = ctx.Attr<float>("epsilon");
    const float residual_alpha = ctx.Attr<float>("residual_alpha");
    auto ln_scales = ctx.MultiInput<phi::DenseTensor>("LnScale");
    auto ln_biases = ctx.MultiInput<phi::DenseTensor>("LnBias");
    NormHelper<T> norm_helper(
        dev_ctx, norm_type, token_num, dim_embed, epsilon, residual_alpha);
    phi::DenseTensor ln_mean, ln_var;
    ln_mean.Resize({{token_num}});
    auto *ln_mean_data =
        dev_ctx.Alloc<U>(&ln_mean, ln_mean.numel() * sizeof(U));
    ln_var.Resize({{token_num}});
    auto *ln_var_data = dev_ctx.Alloc<U>(&ln_var, ln_var.numel() * sizeof(U));

    // 2. qkv
    // x: qkv's input [batch_size, seq_len, dim_embed]
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    auto qkv_weights = ctx.MultiInput<phi::DenseTensor>("QKVW");
    auto qkv_biases = ctx.MultiInput<phi::DenseTensor>("QKVBias");
    const bool trans_qkvw = ctx.Attr<bool>("trans_qkvw");
    const auto qkv_w_dims = qkv_weights[0]->dims();
    int num_head = trans_qkvw ? qkv_w_dims[1] : qkv_w_dims[2];
    int dim_head = trans_qkvw ? qkv_w_dims[2] : qkv_w_dims[3];
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    auto cache_k_scale = ctx.Attr<std::vector<float>>("cache_k_scale");
    auto cache_v_scale = ctx.Attr<std::vector<float>>("cache_v_scale");
    auto cache_k_out_scale = ctx.Attr<std::vector<float>>("cache_k_out_scale");
    auto cache_v_out_scale = ctx.Attr<std::vector<float>>("cache_v_out_scale");
    bool do_cachekv_quant = (cache_k_scale.size() != 0);

    auto quant_round_type = ctx.Attr<int>("quant_round_type");
    auto quant_max_bound = ctx.Attr<float>("quant_max_bound");
    auto quant_min_bound = ctx.Attr<float>("quant_min_bound");

    // Set a flag whether need to add Matmul / Layernorm bias.
    bool compute_bias = qkv_biases.size() > 0;
    bool compute_ln_bias = ln_biases.size() > 0;

    // (transA, transB, compute_bias) = (false, trans_qkvw, false)
    // Since we fused QKVBias into QKVBiasAddTransposeSplit kernel, here we
    // set compute_bias as false.

    // auto mixed_gemm_runner = paddle::operators::CutlassFpAIntBGemmRunner<
    //     typename PDDataTypeTraits<T>::DataType,
    //     uint8_t>();
    auto qkv_compute = GEMMHelper<T>(
        dev_ctx, token_num, output_size, input_size, "None", trans_qkvw);

    phi::DenseTensor qkv_out;
    qkv_out.Resize({{token_num, 3, num_head, dim_head}});
    auto *qkv_out_data =
        dev_ctx.Alloc<T>(&qkv_out, qkv_out.numel() * sizeof(T));

    // 2.1 rotary
    auto *rotary_tensor = ctx.Input<phi::DenseTensor>("RotaryPosEmb");
    const int rotary_emb_dims = ctx.Attr<int>("rotary_emb_dims");

    // 3. fmha
    AttnDropoutParam attn_param(
        true, "upscale_in_train", 0.0, true, true, 0, nullptr);
    auto fmha_compute =
        FMHARef<T>(dev_ctx, bsz, seq_len, num_head, dim_head, attn_param);
    auto *src_mask = ctx.Input<phi::DenseTensor>("SrcMask");
    auto cache_kvs = ctx.MultiInput<phi::DenseTensor>("CacheKV");
    auto cache_kv_outs = ctx.MultiOutput<phi::DenseTensor>("CacheKVOut");
    // auto *time_step = ctx.Input<phi::DenseTensor>("TimeStep");
    auto pre_caches = ctx.MultiInput<phi::DenseTensor>("PreCaches");
    int cache_offset = 0;
    if (pre_caches.size() > 0) {
      cache_offset = pre_caches[0]->dims()[3];
    }

    auto out_seq_len = seq_len;
    if (time_step) {
      PADDLE_ENFORCE_EQ(time_step->place(),
                        platform::CPUPlace(),
                        platform::errors::PreconditionNotMet(
                            "The place of input(TimeStep) must be CPUPlace."));
      // cache_seq_len
      int time_step_value = time_step->data<int>()[0];
      PADDLE_ENFORCE_GT(time_step_value,
                        0,
                        platform::errors::PreconditionNotMet(
                            "The value of time_step must > 0, but now is %d",
                            time_step_value));
      PADDLE_ENFORCE_EQ(
          seq_len,
          1,
          platform::errors::PreconditionNotMet(
              "In decode stage, the seq_len of input must be 1, but now is %d",
              seq_len));
      out_seq_len += time_step_value;
    } else {
      out_seq_len += cache_offset;
    }

    // whether to broadcast 2nd dimension for src_mask, default true
    // if mask_broadcast_num_heads if False, which means src_mask shape
    // will be:
    // 1. [batch_size, num_head, seq_len, seq_len] for encoder
    // 2. [batch_size, num_heads, 1, time_step+1] for decoder
    // and do not need to broadcast num_heads dimension when calculating
    // attn_mask offset in MHA
    bool mask_broadcast_num_heads = true;
    if (src_mask) {
      if (src_mask->dims()[1] == 1) {
        mask_broadcast_num_heads = true;
      } else if (src_mask->dims()[1] == num_head) {
        mask_broadcast_num_heads = false;
      } else {
        PADDLE_THROW(platform::errors::InvalidArgument(
            "Unknow dimension for attn_mask, the num_head(2nd) "
            "dimension is invalid, it should be 1 or num_head(%d), "
            "but got %d",
            num_head,
            src_mask->dims()[1]));
      }
    }

    phi::DenseTensor q_transpose_out, kv_transpose_out, qk_out;
    q_transpose_out.Resize({{bsz, num_head, seq_len, dim_head}});
    auto *q_transpose_out_data =
        dev_ctx.Alloc<T>(&q_transpose_out, q_transpose_out.numel() * sizeof(T));

    kv_transpose_out.Resize({{2, bsz, num_head, seq_len, dim_head}});
    auto *kv_transpose_out_data = dev_ctx.Alloc<T>(
        &kv_transpose_out, kv_transpose_out.numel() * sizeof(T));

    if (encoder_remove_padding) {
      InitValue(dev_ctx,
                q_transpose_out_data,
                q_transpose_out.numel(),
                static_cast<T>(0.));
      InitValue(dev_ctx,
                kv_transpose_out_data,
                kv_transpose_out.numel(),
                static_cast<T>(0.));
    }

    if (FLAGS_fmha_mode == "naive") {
      qk_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *qk_out_data = dev_ctx.Alloc<T>(&qk_out, qk_out.numel() * sizeof(T));
    }

    phi::DenseTensor src_mask_out;
    if (FLAGS_fmha_mode == "naive") {
      if (cache_offset > 0) {
        src_mask_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
        auto *src_mask_out_data =
            dev_ctx.Alloc<T>(&src_mask_out, src_mask_out.numel() * sizeof(T));
      }
    }

    // [2, bs, num_head, cache_seq_len + seq_len, head_dim]
    phi::DenseTensor pre_cache_kv_out;
    if (cache_offset > 0) {
      pre_cache_kv_out.Resize(
          {{2, bsz, num_head, seq_len + cache_offset, dim_head}});
      auto *pre_cache_kv_out_data = dev_ctx.Alloc<T>(
          &pre_cache_kv_out, pre_cache_kv_out.numel() * sizeof(T));
    }

    phi::DenseTensor softmax_out;
    phi::DenseTensor attn_dropout_mask_out, attn_dropout_out;
    phi::DenseTensor qktv_out, fmha_out;
    if (FLAGS_fmha_mode == "naive") {
      softmax_out.Resize({{bsz, num_head, seq_len, out_seq_len}});
      auto *softmax_out_data =
          dev_ctx.Alloc<T>(&softmax_out, softmax_out.numel() * sizeof(T));
    }

    // unpadding_q/unpadding_k/unpadding_v: [token_num, num_head, dim_head]
    phi::DenseTensor unpadding_q, unpadding_k, unpadding_v;
    phi::DenseTensor softmax_lse, seed_offset;
    if (FLAGS_fmha_mode == "flash_attention_v2" && encoder_remove_padding) {
      unpadding_q.Resize({{token_num, num_head, dim_head}});
      unpadding_k.Resize({{token_num, num_head, dim_head}});
      unpadding_v.Resize({{token_num, num_head, dim_head}});
      cu_seqlens_k.Resize(cu_seqlens_q.dims());

      dev_ctx.Alloc<T>(&unpadding_q, unpadding_q.numel() * sizeof(T));
      dev_ctx.Alloc<T>(&unpadding_k, unpadding_k.numel() * sizeof(T));
      dev_ctx.Alloc<T>(&unpadding_v, unpadding_v.numel() * sizeof(T));
      dev_ctx.Alloc<int32_t>(&cu_seqlens_k,
                             cu_seqlens_k.numel() * sizeof(int32_t));
    }

    T *attn_dropout_mask_out_data = nullptr;
    T *attn_dropout_data_data = nullptr;

    qktv_out.Resize({{bsz, num_head, seq_len, dim_head}});
    auto *qktv_out_data =
        dev_ctx.Alloc<T>(&qktv_out, qktv_out.numel() * sizeof(T));
    if (remove_padding) {
      fmha_out.Resize({{token_num, num_head, dim_head}});
    } else {
      fmha_out.Resize({{bsz, seq_len, num_head, dim_head}});
    }
    auto *fmha_out_data =
        dev_ctx.Alloc<T>(&fmha_out, fmha_out.numel() * sizeof(T));
    if (FLAGS_fmha_mode != "flash_attention_v2") {
      if (remove_padding && time_step) {
        InitValue(dev_ctx, fmha_out_data, fmha_out.numel(), static_cast<T>(0.));
      }
    }

    // 4. out_linear
    auto out_linear_weights = ctx.MultiInput<phi::DenseTensor>("OutLinearW");
    auto out_linear_biases = ctx.MultiInput<phi::DenseTensor>("OutLinearBias");
    int ring_id = ctx.Attr<int>("ring_id");
    auto *custom_comm = GetCustomNCCLComm(dev_ctx, ring_id);
    // (transA, transB, compute_bias) = (false, false, false)

    auto out_linear_compute = GEMMHelper<T>(
        dev_ctx, token_num, dim_embed, hidden_size, "None", false);

    // 5. ln(residual + bias)
    auto ffn_ln_scales = ctx.MultiInput<phi::DenseTensor>("FFNLnScale");
    auto ffn_ln_biases = ctx.MultiInput<phi::DenseTensor>("FFNLnBias");
    phi::DenseTensor bias_dropout_residual_out, dropout_mask_out;
    T *bias_dropout_residual_out_data = nullptr;
    if (pre_layer_norm) {
      bias_dropout_residual_out.Resize({{token_num, dim_embed}});
      bias_dropout_residual_out_data =
          dev_ctx.Alloc<T>(&bias_dropout_residual_out,
                           bias_dropout_residual_out.numel() * sizeof(T));
    }
    uint8_t *dropout_mask_out_data = nullptr;

    // 6. ffn matmul1
    auto ffn1_weights = ctx.MultiInput<phi::DenseTensor>("FFN1Weight");
    auto ffn1_biases = ctx.MultiInput<phi::DenseTensor>("FFN1Bias");
    auto ffn1_weight_dim = ffn1_weights[0]->dims();
    // if quant weight,
    // matmul weight is transposed
    int dim_ffn = ffn1_weight_dim[1];
    FFNHelper<T> ffn1_helper(
        dev_ctx, act_method, token_num, dim_ffn, dim_embed, "None");

    phi::DenseTensor ffn1_out;
    ffn1_out.Resize({{token_num, dim_ffn}});
    auto *ffn1_out_data =
        dev_ctx.Alloc<T>(&ffn1_out, ffn1_out.numel() * sizeof(T));

    // Note(Zhengzekang): It is no need when using FP16 matmul.
    phi::DenseTensor mixgemm_workspace;
    char *mixgemm_workspace_data = nullptr;

    // 7. ffn act + bias
    DropoutParam ffn1_dropout_param(true, 0, true, true, 0.0, nullptr, 0);
    FusedDropoutHelper<T, int8_t> fused_act_dropout_helper(
        dev_ctx, token_num, dim_ffn, ffn1_dropout_param);
    phi::DenseTensor ffn1_dropout_out, ffn1_dropout_mask;
    int tmp_dim_ffn = dim_ffn;
    if (use_glu) tmp_dim_ffn /= 2;
    int8_t *ffn1_dropout_mask_data = nullptr;
    ffn1_dropout_out.Resize({{token_num, tmp_dim_ffn}});
    auto *ffn1_dropout_out_data = dev_ctx.Alloc<T>(
        &ffn1_dropout_out, ffn1_dropout_out.numel() * sizeof(T));

    // 8. ffn2 matmul
    auto ffn2_weights = ctx.MultiInput<phi::DenseTensor>("FFN2Weight");
    auto ffn2_biases = ctx.MultiInput<phi::DenseTensor>("FFN2Bias");
    auto ffn2_linear_compute = GEMMHelper<T>(
        dev_ctx, token_num, dim_embed, tmp_dim_ffn, "None", false);

    // 9. ffn2 residual bias
    DropoutParam ffn2_dropout_param(true, 0, true, true, 0.0, nullptr, 0);
    FusedDropoutLayerNormHelper<T, uint8_t> ffn2_fused_dropout_helper(
        dev_ctx,
        token_num,
        dim_embed,
        ffn2_dropout_param,
        epsilon,
        residual_alpha);

    phi::DenseTensor tmp_out, tmp_out_rm_padding;
    tmp_out.Resize({{token_num, dim_embed}});
    if (encoder_remove_padding) {
      tmp_out_rm_padding.Resize({{token_num, dim_embed}});
      auto *tmp_out_rm_padding_data = dev_ctx.Alloc<T>(
          &tmp_out_rm_padding, tmp_out_rm_padding.numel() * sizeof(T));
    }
    auto *tmp_out_data =
        dev_ctx.Alloc<T>(&tmp_out, tmp_out.numel() * sizeof(T));

    const T *x_data;
    if (encoder_remove_padding) {
      x_data = x_remove_padding.data<T>();
    } else {
      x_data = input_x->data<T>();
    }
    phi::DenseTensor *buf0 = nullptr;
    phi::DenseTensor *buf1 = nullptr;

    // step0:  x   --> buf1
    // step1: buf1 --> buf0
    // step2: buf0 --> buf1
    int layers = qkv_weights.size();
    if (encoder_remove_padding) {
      // In the case of variable lengths, the padding needs to be rebuilt
      // eventually. So buf0 and buf1 do not need to be changed according to the
      // pre_layer_norm and the number of layers.
      buf0 = &tmp_out;
      buf1 = &tmp_out_rm_padding;
    } else {
      if (pre_layer_norm) {
        if (layers & 1) {
          // odd, set buf1 as out
          buf0 = &tmp_out;
          buf1 = out;
        } else {
          // even, set buf0 as out
          buf0 = out;
          buf1 = &tmp_out;
        }
      } else {
        buf0 = &tmp_out;
        buf1 = out;
      }
    }
    auto start = std::chrono::system_clock::now();

    VLOG(1) << "input_x->" << input_x->dims();

    VLOG(1) << "input_x " << *input_x;

    for (int i = 0; i < layers; ++i) {
      // step1. layer_norm
      if (i == 0 && pre_layer_norm) {
        norm_helper.Norm(x_data,
                         ln_scales[i],
                         compute_ln_bias ? ln_biases[i] : nullptr, /*norm_bias*/
                         &ln_mean,                                 /*mean*/
                         &ln_var,                                  /*var*/
                         buf1);
      }

#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step1";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "ln1_out:" << *buf1;
#endif
#endif

      // step2. qkv
      // NOTE: In decoder stage, bias is fused in fmha. In encoder stage, bias
      // is fused in QKVBiasAddTransposeSplit
      const phi::DenseTensor *qkv_bias =
          qkv_biases.size() > 0 ? qkv_biases[i] : nullptr;
      if (!pre_layer_norm && i == 0) {
        const phi::DenseTensor *tmp_input_x =
            (encoder_remove_padding) ? &x_remove_padding : input_x;
        VLOG(5) << "Doing !pre_layer_norm&&i==0, qkv gemm, mnk:" << token_num
                << ", " << output_size << ", " << input_size;
        qkv_compute.Compute(tmp_input_x,
                            qkv_weights[i],
                            /*weight_scale*/ nullptr,
                            /*bias*/ nullptr,
                            &mixgemm_workspace,
                            &qkv_out);
      } else {
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
        VLOG(2) << "qkv_weights:" << *(qkv_weights[i]);
#endif
#endif
        VLOG(5) << "Doing qkv gemm, mnk:" << token_num << ", " << output_size
                << ", " << input_size;
        qkv_compute.Compute(buf1,
                            qkv_weights[i],
                            /*weight_scale*/ nullptr,
                            /*bias*/ nullptr,
                            &mixgemm_workspace,
                            &qkv_out);
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step2";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "qkv_out:" << qkv_out;
#endif
#endif

      // 2. cache kv
      auto write_cache_kv_helper = WriteCacheKVHelper<T>(
          dev_ctx, quant_round_type, quant_max_bound, quant_min_bound);

      // step3. fmha
      const phi::DenseTensor *cache_kv =
          cache_kvs.size() > 0 ? cache_kvs[i] : nullptr;
      phi::DenseTensor *cache_kv_out = cache_kv ? cache_kv_outs[i] : nullptr;
      int cache_bsz = 0;
      if (cache_kv) {
        cache_bsz = cache_kv->dims()[1];
      }

      if (time_step) {  // generation decoder stage
        // [2, batch_size, num_head, max_seq_len, head_size]
        int max_seq_len = cache_kv->dims()[3];
        fmha<T>(dev_ctx,
                qkv_out,
                *qkv_bias,
                src_mask,
                nullptr,
                sequence_lengths,
                rotary_tensor,
                beam_cache_offset,
                cache_kv_out,
                &fmha_out,
                bsz,
                cache_bsz,
                seq_len,
                max_seq_len,
                num_head,
                dim_head,
                src_mask->dims()[3] - 1,
                rotary_emb_dims,
                1. / sqrt(dim_head),
                mask_broadcast_num_heads,
                compute_bias,
                use_neox_rotary_style,
                nullptr,  // qkv_out_scale
                nullptr,  // out_linear_shift
                nullptr,  // out_smooth_shift
                (do_cachekv_quant) ? cache_k_scale[i] : -1.0,
                (do_cachekv_quant) ? cache_v_scale[i] : -1.0,
                (do_cachekv_quant) ? cache_k_out_scale[i] : -1.0,
                (do_cachekv_quant) ? cache_v_out_scale[i] : -1.0);
      } else if (cache_kv_out) {  // generation context stage
        const phi::DenseTensor *pre_cache_kv_tensor =
            pre_caches.size() > 0 ? pre_caches[i] : nullptr;
        phi::DenseTensor *pre_cache_kv_out_tmp =
            cache_offset > 0 ? &pre_cache_kv_out : nullptr;
        phi::DenseTensor *src_mask_tmp =
            cache_offset > 0 ? &src_mask_out : nullptr;
        const int *sequence_lengths_data =
            encoder_remove_padding ? sequence_lengths->data<int>() : nullptr;
        qkv_bias_add_transpose_split<T>(
            dev_ctx,
            q_transpose_out_data,
            kv_transpose_out_data,
            qkv_out_data,
            qkv_bias ? qkv_bias->data<T>() : nullptr,
            padding_offset_data,
            token_num,
            bsz,
            num_head,
            seq_len,
            dim_head,
            compute_bias);

        // q_transpose_out_data [bs, head_num, seq_len, dim_head]
        // kv_transpose_out_data [2， bs, head_num, seq_len, dim_head]
        if (rotary_emb_dims != 0) {
          auto *rotary_emb_data = rotary_tensor->data<float>();
          const int *sequence_lengths_data =
              encoder_remove_padding ? sequence_lengths->data<int>() : nullptr;
          rotary_qk(dev_ctx,
                    q_transpose_out_data,
                    kv_transpose_out_data,
                    q_transpose_out_data,
                    kv_transpose_out_data,
                    rotary_emb_data,
                    sequence_lengths_data,
                    rotary_emb_dims,
                    rotary_tensor->dims()[1],
                    bsz,
                    num_head,
                    seq_len,
                    dim_head,
                    use_neox_rotary_style);
        }

        phi::DenseTensor *tmp_padding_offset_tensor =
            encoder_remove_padding ? &padding_offset_tensor : nullptr;

        if (FLAGS_fmha_mode == "flash_attention_v2") {
          if (!encoder_remove_padding) {
            PADDLE_THROW(paddle::platform::errors::Unimplemented("encoder_remove_padding must be true when fmha_mode is flash_attention_v2"));
          }
          TransposeSplit<T>(dev_ctx,
                            unpadding_q.data<T>(),
                            unpadding_k.data<T>(),
                            unpadding_v.data<T>(),
                            q_transpose_out.data<T>(),
                            kv_transpose_out.data<T>(),
                            padding_offset_data,
                            sequence_lengths->data<int>(),
                            token_num,
                            bsz,
                            num_head,
                            seq_len,
                            dim_head);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "TransposeSplit";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "unpadding_q:" << unpadding_q;
      VLOG(2) << "unpadding_k:" << unpadding_k;
      VLOG(2) << "unpadding_v:" << unpadding_v;
#endif
#endif                   
          phi::Copy(dev_ctx,
                    cu_seqlens_q,
                    cu_seqlens_k.place(),
                    false,
                    &cu_seqlens_k);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "Copy";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "cu_seqlens_q:" << cu_seqlens_q;
      VLOG(2) << "cu_seqlens_k:" << cu_seqlens_k;
#endif
#endif  

          // fmha_out[token_num, num_head, dim_head]
          phi::FlashAttnUnpaddedKernel<T>(dev_ctx,
                                          unpadding_q,
                                          unpadding_k,
                                          unpadding_v,
                                          cu_seqlens_q,
                                          cu_seqlens_k,
                                          none /*fixed_seed_offset*/,
                                          none /*attn_mask*/,
                                          seq_len,
                                          seq_len,
                                          1.0f / sqrt(float(dim_head)),
                                          0.0,
                                          true /*causal*/,
                                          false,
                                          true /* is_test*/,
                                          "" /*rng_name*/,
                                          &fmha_out,
                                          &softmax_out,
                                          &softmax_lse,
                                          &seed_offset);
        } else {
          fmha_compute.Compute(pre_cache_kv_tensor,
                               src_mask,
                               tmp_padding_offset_tensor,
                               sequence_lengths,
                               &q_transpose_out,
                               &kv_transpose_out,
                               pre_cache_kv_out_tmp,
                               &qk_out,
                               src_mask_tmp,
                               &softmax_out,
                               &attn_dropout_mask_out,
                               &attn_dropout_out,
                               &qktv_out,
                               &fmha_out,
                               token_num,
                               mask_broadcast_num_heads);
        }

        write_cache_kv_helper.Compute(
            &pre_cache_kv_out,
            cache_kv_out,       // int8_t
            &kv_transpose_out,  // T
            sequence_lengths_data,
            cache_bsz,
            bsz,
            num_head,
            seq_len,
            dim_head,
            cache_offset,
            (do_cachekv_quant) ? cache_k_scale[i] : -1.0,
            (do_cachekv_quant) ? cache_v_scale[i] : -1.0);

      } else {  // not generation
        // TODO(wangxi): can remove dropout in inference
        qkv_bias_add_transpose_split<T>(
            dev_ctx,
            q_transpose_out_data,
            kv_transpose_out_data,
            qkv_out_data,
            qkv_bias ? qkv_bias->data<T>() : nullptr,
            padding_offset_data,
            token_num,
            bsz,
            num_head,
            seq_len,
            dim_head,
            compute_bias);

        // q_transpose_out_data [bs, head_num, seq_len, dim_head]
        // kv_transpose_out_data [2， bs, head_num, seq_len, dim_head]
        if (rotary_emb_dims != 0) {
          auto *rotary_emb_data = rotary_tensor->data<float>();
          const int *sequence_lengths_data =
              encoder_remove_padding ? sequence_lengths->data<int>() : nullptr;
          rotary_qk(dev_ctx,
                    q_transpose_out_data,
                    kv_transpose_out_data,
                    q_transpose_out_data,
                    kv_transpose_out_data,
                    rotary_emb_data,
                    sequence_lengths_data,
                    rotary_emb_dims,
                    rotary_tensor->dims()[1],
                    bsz,
                    num_head,
                    seq_len,
                    dim_head,
                    use_neox_rotary_style);
        }
        phi::DenseTensor *tmp_padding_offset_tensor =
            encoder_remove_padding ? &padding_offset_tensor : nullptr;

        if (FLAGS_fmha_mode == "flash_attention_v2") {
          if (!encoder_remove_padding) {
            PADDLE_THROW(paddle::platform::errors::Unimplemented("encoder_remove_padding must be true when fmha_mode is flash_attention_v2"));
          }
          TransposeSplit<T>(dev_ctx,
                            unpadding_q.data<T>(),
                            unpadding_k.data<T>(),
                            unpadding_v.data<T>(),
                            q_transpose_out.data<T>(),
                            kv_transpose_out.data<T>(),
                            padding_offset_data,
                            sequence_lengths->data<int>(),
                            token_num,
                            bsz,
                            num_head,
                            seq_len,
                            dim_head);
          phi::Copy(dev_ctx,
                    cu_seqlens_q,
                    cu_seqlens_k.place(),
                    false,
                    &cu_seqlens_k);

          // fmha_out[token_num, num_head, dim_head]
          phi::FlashAttnUnpaddedKernel<T>(dev_ctx,
                                          unpadding_q,
                                          unpadding_k,
                                          unpadding_v,
                                          cu_seqlens_q,
                                          cu_seqlens_k,
                                          none /*fixed_seed_offset*/,
                                          none /*attn_mask*/,
                                          seq_len,
                                          seq_len,
                                          1.0f / sqrt(float(dim_head)),
                                          0.0,
                                          true /*causal*/,
                                          false,
                                          true /* is_test*/,
                                          "" /*rng_name*/,
                                          &fmha_out,
                                          &softmax_out,
                                          &softmax_lse,
                                          &seed_offset);
        } else {
          fmha_compute.Compute(cache_kv,
                               src_mask,
                               tmp_padding_offset_tensor,
                               sequence_lengths,
                               &q_transpose_out,
                               &kv_transpose_out,
                               cache_kv_out,
                               &qk_out,
                               nullptr,
                               &softmax_out,
                               &attn_dropout_mask_out,
                               &attn_dropout_out,
                               &qktv_out,
                               &fmha_out,
                               token_num,
                               mask_broadcast_num_heads);
        }
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step3";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "fmha_out:" << fmha_out;
#endif
#endif
      VLOG(5) << "Doing out_linear gemm, mnk:" << token_num << ", " << dim_embed
              << ", " << hidden_size;
      if (pre_layer_norm) {
        if (custom_comm) {
          custom_comm->SwapInput(buf1);
        }

        out_linear_compute.Compute(&fmha_out,
                                   out_linear_weights[i],
                                   /*weight_scale*/ nullptr,
                                   /*bias*/ nullptr,
                                   &mixgemm_workspace,
                                   buf1);

        if (custom_comm) {
          *buf1 = custom_comm->AllReduce();
        } else {
          VLOG(1) << "ALLREDUCE " << buf1->numel();
          AllReduce<T>(*buf1, ring_id, buf1->numel(), dev_ctx);
        }
      } else {
        if (custom_comm) {
          custom_comm->SwapInput(buf0);
        }
        out_linear_compute.Compute(&fmha_out,
                                   out_linear_weights[i],
                                   /*weight_scale*/ nullptr,
                                   /*bias*/ nullptr,
                                   &mixgemm_workspace,
                                   buf0);

        if (custom_comm) {
          *buf0 = custom_comm->AllReduce();
        } else {
          AllReduce<T>(*buf0, ring_id, buf0->numel(), dev_ctx);
        }
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step4";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "out_linear_out:" << *buf1;
#endif
#endif

      // step5. ln(residual + dropout(input + bias))
      if (pre_layer_norm) {
        norm_helper.NormResidualBias(
            buf1->data<T>(),
            x_data,
            compute_bias ? out_linear_biases[i] : nullptr, /*skip_bias*/
            ffn_ln_scales[i],
            compute_ln_bias ? ffn_ln_biases[i] : nullptr, /*norm_bias*/
            &ln_mean,                                     /*mean*/
            &ln_var,                                      /*var*/
            &bias_dropout_residual_out,
            buf1);
      } else {
        auto *residual_data = (i == 0 ? x_data : buf1->data<T>());
        norm_helper.NormResidualBias(
            buf0->data<T>(),
            residual_data,
            compute_bias ? out_linear_biases[i] : nullptr, /*skip_bias*/
            ln_scales[i],
            compute_ln_bias ? ln_biases[i] : nullptr, /*norm_bias*/
            &ln_mean,                                 /*mean*/
            &ln_var,                                  /*var*/
            buf0,
            buf1);
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step5";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "ffn1_input:" << *buf1;
#endif
#endif
      // step6. ffn matmul1
      ffn1_helper.Compute(buf1,
                          ffn1_weights[i],
                          /*weight_scale*/ nullptr,
                          compute_bias ? ffn1_biases[i] : nullptr,
                          &mixgemm_workspace,
                          &ffn1_out,
                          &ffn1_dropout_out);
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step6";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "ffn1_output:" << ffn1_out;
#endif
#endif

      // step7. ffn2 matmul
      if (pre_layer_norm) {
        if (custom_comm) {
          custom_comm->SwapInput(buf1);
        }
        ffn2_linear_compute.Compute(&ffn1_dropout_out,
                                    ffn2_weights[i],
                                    nullptr,
                                    /*bias*/ nullptr,
                                    &mixgemm_workspace,
                                    buf1);
      } else {
        if (custom_comm) {
          custom_comm->SwapInput(buf0);
        }
        ffn2_linear_compute.Compute(&ffn1_dropout_out,
                                    ffn2_weights[i],
                                    nullptr,
                                    /*bias*/ nullptr,
                                    &mixgemm_workspace,
                                    buf0);
      }
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step8.0";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      if (pre_layer_norm) {
        VLOG(2) << "ffn2_out, buf1:" << *buf1;
      } else {
        VLOG(2) << "ffn2_out, buf0:" << *buf0;
      }
#endif
#endif

      if (pre_layer_norm) {
        VLOG(4) << "MPAllReduce 4: " << buf1->numel();
        if (custom_comm) {
          *buf1 = custom_comm->AllReduce();
        } else {
          VLOG(1) << "ALLREDUCE ffn" << buf1->numel();

          auto ar_start = std::chrono::system_clock::now();
          AllReduce<T>(*buf1, ring_id, buf1->numel(), dev_ctx);
          auto ar_end = std::chrono::system_clock::now();
          auto ar_duration =
              std::chrono::duration_cast<std::chrono::microseconds>(ar_end -
                                                                    ar_start);
          VLOG(3) << "reduce elapse "
                  << double(ar_duration.count()) *
                         std::chrono::microseconds::period::num /
                         std::chrono::microseconds::period::den
                  << " SEC";
        }
      } else {
        VLOG(4) << "MPAllReduce 4: " << buf0->numel();
        if (custom_comm) {
          *buf0 = custom_comm->AllReduce();
        } else {
          AllReduce<T>(*buf0, ring_id, buf0->numel(), dev_ctx);
        }
      }

#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step8.1";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      if (pre_layer_norm) {
        VLOG(2) << "ffn2_out_rd:" << *buf1;
      } else {
        VLOG(2) << "ffn2_out_rd:" << *buf0;
      }
#endif
#endif

      // step8. residual bias
      // TODO(wangxi): remove dropout mask in inference
      if (pre_layer_norm) {
        // TODO(wangxi): remove dropout mask in inference
        if (i < layers - 1) {
          norm_helper.NormResidualBias(
              buf1->data<T>(),
              bias_dropout_residual_out_data,
              compute_bias ? ffn2_biases[i] : nullptr, /*skip_bias*/
              ln_scales[i + 1],
              compute_ln_bias ? ln_biases[i + 1] : nullptr, /*norm_bias*/
              &ln_mean,                                     /*mean*/
              &ln_var,                                      /*var*/
              buf1,
              buf0);
        } else {
          ffn2_fused_dropout_helper.ResidualDropoutBias(
              dev_ctx,
              buf1->data<T>(),
              bias_dropout_residual_out_data,
              compute_bias ? ffn2_biases[i]->data<T>() : nullptr,
              buf1->data<T>(),
              dropout_mask_out_data);
        }
      } else {
        norm_helper.NormResidualBias(
            buf0->data<T>(),
            buf1->data<T>(),
            compute_bias ? ffn2_biases[i] : nullptr, /*skip_bias*/
            ffn_ln_scales[i],
            compute_ln_bias ? ffn_ln_biases[i] : nullptr, /*norm_bias*/
            &ln_mean,                                     /*mean*/
            &ln_var,                                      /*var*/
            buf0,
            buf1);
      }

#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER
      VLOG(2) << "step9";
#ifdef _DEBUG_FUSED_MULTI_TRANSFORMER_PRINT_TENSOR
      VLOG(2) << "residual_out:" << *buf1;
#endif
#endif
      if (pre_layer_norm) {
        x_data = buf1->data<T>();
        std::swap(buf0, buf1);
      }
    }
    auto end = std::chrono::system_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    VLOG(3) << "ELAPSE "
            << double(duration.count()) *
                   std::chrono::microseconds::period::num /
                   std::chrono::microseconds::period::den
            << " SEC";
    if (encoder_remove_padding) {
      if (pre_layer_norm) {
        InvokeRebuildPadding(dev_ctx,
                             from_data,
                             buf0->data<T>(),
                             padding_offset_data,
                             token_num,
                             dim_embed);
      } else {
        InvokeRebuildPadding(dev_ctx,
                             from_data,
                             buf1->data<T>(),
                             padding_offset_data,
                             token_num,
                             dim_embed);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

#if CUDA_VERSION >= 11000
PD_REGISTER_STRUCT_KERNEL(fused_multi_transformer,
                          GPU,
                          ALL_LAYOUT,
                          ops::FusedMultiTransformerOpKernel,
                          float,
                          plat::float16,
                          plat::bfloat16) {}
#else
PD_REGISTER_STRUCT_KERNEL(fused_multi_transformer,
                          GPU,
                          ALL_LAYOUT,
                          ops::FusedMultiTransformerOpKernel,
                          float,
                          plat::float16) {}
#endif   
