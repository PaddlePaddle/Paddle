/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hip/hip_fp16.h>

#include <hipcub/hipcub.hpp>

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/fused/attention_layer_norm.h"
#include "paddle/fluid/operators/fused/attn_gemm.h"
#include "paddle/fluid/operators/fused/fmha_ref.h"
#include "paddle/fluid/operators/fused/fused_dropout_helper.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/phi/api/include/tensor.h"
#include "paddle/phi/backends/gpu/gpu_device_function.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/funcs/functors.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/transpose_function.cu.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/process_group_nccl.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif

namespace paddle {
namespace operators {

template <typename T>
static void AllReduce(phi::DenseTensor &tensor,  // NOLINT
                      const int ring_id,
                      const phi::GPUContext &ctx) {
  if (ring_id == -1) return;
#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();

  if (map->has(ring_id)) {
    paddle::distributed::ProcessGroup *pg = map->get(ring_id);
    auto pg_nccl = static_cast<distributed::ProcessGroupNCCL *>(pg);
    paddle::distributed::AllreduceOptions opts;
    opts.reduce_op = distributed::ReduceOp::SUM;
    auto task = pg_nccl->AllReduce(&tensor, tensor, opts, true, true);
    task->Wait();
  } else {
    auto dtype = platform::ToNCCLDataType(
        framework::TransToProtoVarType(tensor.dtype()));
    int64_t numel = tensor.numel();
    const void *sendbuff = tensor.data<T>();
    auto place = ctx.GetPlace();
    void *recvbuff = ctx.template Alloc<T>(&tensor, tensor.numel() * sizeof(T));
    auto comm = platform::NCCLCommContext::Instance().Get(ring_id, place);
    auto stream = ctx.stream();
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
        sendbuff, recvbuff, numel, dtype, ncclSum, comm->comm(), stream));
  }
#else
  PADDLE_THROW(platform::errors::Unimplemented(
      "PaddlePaddle should compile with NCCL or RCCL when used tensor model "
      "parallel op."));
#endif
}

template <typename T>
class FusedAttentionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto *input_x = ctx.Input<phi::DenseTensor>("X");
    auto &dev_ctx = ctx.template device_context<phi::GPUContext>();
    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *ln_scale = ctx.Input<phi::DenseTensor>("LnScale");
    auto *ln_bias = ctx.Input<phi::DenseTensor>("LnBias");
    auto *ln_mean = ctx.Output<phi::DenseTensor>("LnMean");
    auto *ln_var = ctx.Output<phi::DenseTensor>("LnVariance");
    auto *ln_out = ctx.Output<phi::DenseTensor>("LnOut");

    const auto num_heads = ctx.Attr<int>("num_heads");
    const auto transpose_qkv_wb = ctx.Attr<bool>("transpose_qkv_wb");

    // x: qkv's input [batch_size, seq_len, dim_embed]
    // if transpose_qkv_wb is False
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    // if transpose_qkv_wb is True
    // y: qkv's weight: [dim_embed, 3 * dim_embed]
    auto *qkv_weight = ctx.Input<phi::DenseTensor>("QKVW");
    auto *qkv_bias = ctx.Input<phi::DenseTensor>("QKVBias");
    auto *qkv_out = ctx.Output<phi::DenseTensor>("QKVOut");
    auto *qkv_bias_out = ctx.Output<phi::DenseTensor>("QKVBiasOut");

    auto *src_mask = ctx.Input<phi::DenseTensor>("SrcMask");
    auto *transpose_out_2 = ctx.Output<phi::DenseTensor>("TransposeOut2");
    auto *cache_kv = ctx.Input<phi::DenseTensor>("CacheKV");
    auto *cache_kv_out = ctx.Output<phi::DenseTensor>("CacheKVOut");
    auto *qk_out = ctx.Output<phi::DenseTensor>("QKOut");
    auto *qktv_out = ctx.Output<phi::DenseTensor>("QKTVOut");
    auto *softmax_out = ctx.Output<phi::DenseTensor>("SoftmaxOut");
    auto *attn_dropout_mask_out =
        ctx.Output<phi::DenseTensor>("AttnDropoutMaskOut");
    auto *attn_dropout_out = ctx.Output<phi::DenseTensor>("AttnDropoutOut");
    auto *src_mask_out = ctx.Output<phi::DenseTensor>("SrcMaskOut");
    auto *fmha_out = ctx.Output<phi::DenseTensor>("FMHAOut");

    auto *out_linear_weight = ctx.Input<phi::DenseTensor>("OutLinearW");
    auto *out_linear_bias = ctx.Input<phi::DenseTensor>("OutLinearBias");
    auto *out_linear_out = ctx.Output<phi::DenseTensor>("OutLinearOut");

    auto *ln_scale_2 = ctx.Input<phi::DenseTensor>("Ln2Scale");
    auto *ln_bias_2 = ctx.Input<phi::DenseTensor>("Ln2Bias");
    auto *dropout_mask_out = ctx.Output<phi::DenseTensor>("DropoutMaskOut");
    auto *bias_dropout_residual_out =
        ctx.Output<phi::DenseTensor>("BiasDropoutResidualOut");
    auto *ln_mean_2 = ctx.Output<phi::DenseTensor>("Ln2Mean");
    auto *ln_var_2 = ctx.Output<phi::DenseTensor>("Ln2Variance");
    const float ln_epsilon = ctx.Attr<float>("ln_epsilon");

    float attn_dropout_rate = ctx.Attr<float>("attn_dropout_rate");
    const bool has_attn_dropout = (attn_dropout_rate != 0.0f);
    DropoutParam dropout_param2(ctx, 0);
    const bool has_dropout = (dropout_param2.dropout_prob != 0.0f);

    bool is_test_1 = ctx.Attr<bool>("is_test");
    auto &dropout_implementation_1 =
        ctx.Attr<std::string>("attn_dropout_implementation");
    bool is_upscale_in_train_1 =
        (dropout_implementation_1 == "upscale_in_train");
    auto *seed_1 =
        ctx.HasInput("Seed1") ? ctx.Input<phi::DenseTensor>("Seed1") : nullptr;
    bool is_fix_seed_1 = ctx.Attr<bool>("attn_dropout_fix_seed");
    int seed_val_1 = ctx.Attr<int>("attn_dropout_seed");
    int ring_id = ctx.Attr<int>("ring_id");

    // final output.
    auto *out = ctx.Output<phi::DenseTensor>("Y");

    // get data ptr for qkv part.
    const auto input_x_dims = input_x->dims();
    const auto qkv_w_dims = qkv_weight->dims();

    auto *x_data = input_x->data<T>();
    auto *qkv_weight_data = qkv_weight->data<T>();
    auto *qkv_bias_data = (qkv_bias == nullptr) ? nullptr : qkv_bias->data<T>();
    auto *qkv_out_data =
        dev_ctx.template Alloc<T>(qkv_out, qkv_out->numel() * sizeof(T));
    auto *qkv_bias_out_data =
        (qkv_bias == nullptr)
            ? nullptr
            : dev_ctx.template Alloc<T>(qkv_bias_out,
                                        qkv_bias_out->numel() * sizeof(T));

    // get data ptr for FMHA.
    auto *transpose_out_2_data = dev_ctx.template Alloc<T>(
        transpose_out_2, transpose_out_2->numel() * sizeof(T));
    auto *cache_kv_out_data =
        (cache_kv_out == nullptr)
            ? nullptr
            : dev_ctx.template Alloc<T>(cache_kv_out,
                                        cache_kv_out->numel() * sizeof(T));
    auto *qk_out_data =
        dev_ctx.template Alloc<T>(qk_out, qk_out->numel() * sizeof(T));
    auto *qktv_out_data =
        dev_ctx.template Alloc<T>(qktv_out, qktv_out->numel() * sizeof(T));
    auto *src_mask_out_data =
        (src_mask == nullptr)
            ? nullptr
            : dev_ctx.template Alloc<T>(src_mask_out,
                                        src_mask_out->numel() * sizeof(T));
    auto *softmax_out_data = dev_ctx.template Alloc<T>(
        softmax_out, softmax_out->numel() * sizeof(T));
    auto *attn_dropout_mask_out_data =
        has_attn_dropout ? dev_ctx.template Alloc<uint8_t>(
                               attn_dropout_mask_out,
                               attn_dropout_mask_out->numel() * sizeof(uint8_t))
                         : nullptr;
    auto *attn_dropout_out_data =
        has_attn_dropout
            ? dev_ctx.template Alloc<T>(attn_dropout_out,
                                        attn_dropout_out->numel() * sizeof(T))
            : nullptr;
    auto *fmha_out_data =
        dev_ctx.template Alloc<T>(fmha_out, fmha_out->numel() * sizeof(T));

    // get data ptr for out_linear.
    auto *out_linear_weight_data = out_linear_weight->data<T>();
    auto *out_linear_bias_data =
        (out_linear_bias == nullptr) ? nullptr : out_linear_bias->data<T>();
    auto *out_linear_out_data = dev_ctx.template Alloc<T>(
        out_linear_out, out_linear_out->numel() * sizeof(T));

    // get data ptr for bias+dropout+residual+layernorm
    auto *dropout_mask_out_data =
        has_dropout
            ? dev_ctx.template Alloc<uint8_t>(
                  dropout_mask_out, dropout_mask_out->numel() * sizeof(uint8_t))
            : nullptr;
    auto *final_out_data =
        dev_ctx.template Alloc<T>(out, out->numel() * sizeof(T));

    int batch_size = input_x_dims[0];
    int max_seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];

    int num_head;
    int dim_head;
    int nranks = 1;
    // get num_head and dim_head in two different ways
    if (!transpose_qkv_wb) {
      num_head = qkv_w_dims[1];
      dim_head = qkv_w_dims[2];
    } else {
      nranks = (qkv_w_dims[0] * 3) / qkv_w_dims[1];
      num_head = num_heads;
      dim_head = dim_embed / (num_head * nranks);
    }

    int bsz_seq = batch_size * max_seq_len;
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    auto layer_norm_compute = AttnLayerNorm<T>(
        ctx.cuda_device_context(), epsilon, bsz_seq, dim_embed);

    bool compute_bias = true;
    if (qkv_bias == nullptr) {
      compute_bias = false;
    }
    // (transA, transB, compute_bias) = (false, true, true)
    bool transB = transpose_qkv_wb ? false : true;
    auto qkv_compute = AttnMatMul<T>(ctx.cuda_device_context(),
                                     false,
                                     transB,
                                     bsz_seq,
                                     output_size,
                                     input_size,
                                     compute_bias);

    AttnDropoutParam attn_dropout_param(is_test_1,
                                        dropout_implementation_1,
                                        attn_dropout_rate,
                                        is_upscale_in_train_1,
                                        is_fix_seed_1,
                                        seed_val_1,
                                        seed_1);
    auto fmha_ref_compute = FMHARef<T>(ctx.cuda_device_context(),
                                       batch_size,
                                       max_seq_len,
                                       num_head,
                                       dim_head,
                                       attn_dropout_param);

    output_size = hidden_size;
    // (transA, transB, compute_bias) = (false, false, false)
    // NOTE(Yuang Liu): For general input size == output size, change the
    // position won't have effects. For mp, the output size is mp_head * dkey
    // which is actually the input size. While the input size is hidden size,
    // which is actually the output size. So for out linear, switch the
    // input size and output size.
    auto out_linear_compute = AttnMatMul<T>(ctx.cuda_device_context(),
                                            false,
                                            false,
                                            bsz_seq,
                                            input_size,
                                            output_size,
                                            false);
    FusedDropoutLayerNormHelper<T, uint8_t> fused_dropout_layernorm_helper(
        ctx.cuda_device_context(),
        bsz_seq,
        dim_embed,
        dropout_param2,
        ln_epsilon);

    if (pre_layer_norm) {
      auto *ln_scale_data =
          (ln_scale == nullptr ? nullptr : ln_scale->data<U>());
      auto *ln_bias_data = (ln_bias == nullptr ? nullptr : ln_bias->data<U>());
      auto *ln_mean_data =
          dev_ctx.template Alloc<U>(ln_mean, ln_mean->numel() * sizeof(U));
      auto *ln_var_data =
          dev_ctx.template Alloc<U>(ln_var, ln_var->numel() * sizeof(U));
      auto *ln_out_data =
          dev_ctx.template Alloc<T>(ln_out, ln_out->numel() * sizeof(T));

      layer_norm_compute.ComputeForward(x_data,
                                        ln_scale_data,
                                        ln_bias_data,
                                        ln_out_data,
                                        ln_mean_data,
                                        ln_var_data);
      qkv_compute.ComputeForward(
          qkv_weight, ln_out, qkv_bias, qkv_out, qkv_bias_out);
    } else {
      qkv_compute.ComputeForward(
          qkv_weight, input_x, qkv_bias, qkv_out, qkv_bias_out);
    }

    if (transpose_qkv_wb) {
      // resize the output for fmha compute
      qkv_out->Resize({batch_size, max_seq_len, 3, num_head, dim_head});
      qkv_bias_out->Resize({batch_size, max_seq_len, 3, num_head, dim_head});
    }

    if (qkv_bias == nullptr) {
      fmha_ref_compute.ComputeForward(*qkv_out,
                                      cache_kv,
                                      src_mask,
                                      transpose_out_2,
                                      cache_kv_out,
                                      qk_out,
                                      src_mask_out,
                                      softmax_out,
                                      attn_dropout_mask_out,
                                      attn_dropout_out,
                                      qktv_out,
                                      fmha_out);
    } else {
      fmha_ref_compute.ComputeForward(*qkv_bias_out,
                                      cache_kv,
                                      src_mask,
                                      transpose_out_2,
                                      cache_kv_out,
                                      qk_out,
                                      src_mask_out,
                                      softmax_out,
                                      attn_dropout_mask_out,
                                      attn_dropout_out,
                                      qktv_out,
                                      fmha_out);
    }

    if (transpose_qkv_wb) {
      // resize the output back to make the shape compatible with infer shape
      qkv_out->Resize({batch_size, max_seq_len, 3 * hidden_size});
      qkv_bias_out->Resize({batch_size, max_seq_len, 3 * hidden_size});
    }

    // fmha_out: [batch_size, seq_len, num_head, head_dim]
    // weight:   [embed_dim, embed_dim]
    // out_linear_out: [batch_size, seq_len, embed_dim]
    out_linear_compute.ComputeForward(
        out_linear_weight, fmha_out, nullptr, out_linear_out, nullptr);
    // tensor model parallel
    AllReduce<T>(*out_linear_out, ring_id, ctx.cuda_device_context());

    bool add_residual = ctx.Attr<bool>("add_residual");
    const T *residual_ptr = add_residual ? x_data : nullptr;
    if (pre_layer_norm) {
      // output = (residual + dropout(input + bias))
      fused_dropout_layernorm_helper.ResidualDropoutBias(
          ctx.cuda_device_context(),
          out_linear_out_data,
          residual_ptr,
          out_linear_bias_data,
          final_out_data,
          dropout_mask_out_data);
    } else {
      // TODO(Xreki): support post layer_norm case when add_residual is false.
      PADDLE_ENFORCE_EQ(add_residual,
                        true,
                        platform::errors::InvalidArgument(
                            "Attribute add_residual is expected to be true "
                            "when pre_layer_norm is false."));

      const U *ln_scale_2_ptr = ln_scale_2 ? ln_scale_2->data<U>() : nullptr;
      const U *ln_bias_2_ptr = ln_bias_2 ? ln_bias_2->data<U>() : nullptr;
      T *bias_dropout_residual_out_ptr = dev_ctx.template Alloc<T>(
          bias_dropout_residual_out,
          bias_dropout_residual_out->numel() * sizeof(T));
      U *ln_mean_2_ptr =
          dev_ctx.template Alloc<U>(ln_mean_2, ln_mean_2->numel() * sizeof(U));
      U *ln_var_2_ptr =
          dev_ctx.template Alloc<U>(ln_var_2, ln_var_2->numel() * sizeof(U));
      // output = layernorm(residual + dropout(input + bias))
      fused_dropout_layernorm_helper.LayernormResidualDropoutBias(
          ctx.cuda_device_context(),
          out_linear_out_data,
          residual_ptr,
          out_linear_bias_data,
          ln_scale_2_ptr,
          ln_bias_2_ptr,
          bias_dropout_residual_out_ptr,
          dropout_mask_out_data,
          final_out_data,
          ln_mean_2_ptr,
          ln_var_2_ptr);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(fused_attention,
                        ops::FusedAttentionOpKernel<float>,
                        ops::FusedAttentionOpKernel<double>,
                        ops::FusedAttentionOpKernel<plat::float16>);
