/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/platform/cuda_device_function.h"

#ifdef PADDLE_WITH_CUDA
#include "paddle/fluid/platform/cudnn_helper.h"
#endif
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/platform/miopen_helper.h"
#endif

#include <hip/hip_fp16.h>
#include "paddle/fluid/operators/elementwise/elementwise_add_op.h"
#include "paddle/fluid/operators/math/math_function.h"

#include "paddle/fluid/operators/fused/fused_attention_op.h"

#include "paddle/fluid/operators/fused/attention_layer_norm.h"
#include "paddle/fluid/operators/fused/attn_gemm.h"
#include "paddle/fluid/operators/fused/fmha_ref.h"
#include "paddle/fluid/operators/fused/fused_dropout_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
class FusedAttentionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto *input_x = ctx.Input<Tensor>("X");

    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *ln_scale = ctx.Input<Tensor>("LnScale");
    auto *ln_bias = ctx.Input<Tensor>("LnBias");
    auto *ln_mean = ctx.Output<Tensor>("LnMean");
    auto *ln_var = ctx.Output<Tensor>("LnVariance");
    auto *ln_out = ctx.Output<Tensor>("LnOut");

    // x: qkv's input [batch_size, seq_len, dim_embed]
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    auto *qkv_weight = ctx.Input<Tensor>("QKVW");
    auto *qkv_bias = ctx.Input<Tensor>("QKVBias");
    auto *qkv_out = ctx.Output<Tensor>("QKVOut");
    auto *qkv_bias_out = ctx.Output<Tensor>("QKVBiasOut");

    // FMHA-ref:
    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto *transpose_out_2 = ctx.Output<Tensor>("TransposeOut2");
    auto *qk_out = ctx.Output<Tensor>("QKOut");
    auto *qktv_out = ctx.Output<Tensor>("QKTVOut");
    auto *softmax_out = ctx.Output<Tensor>("SoftmaxOut");
    auto *attn_dropout_mask_out = ctx.Output<Tensor>("AttnDropoutMaskOut");
    auto *attn_dropout_out = ctx.Output<Tensor>("AttnDropoutOut");
    auto *src_mask_out = ctx.Output<Tensor>("SrcMaskOut");
    auto *fmha_out = ctx.Output<Tensor>("FMHAOut");

    // out_linear
    auto *out_linear_weight = ctx.Input<Tensor>("OutLinearW");
    auto *out_linear_bias = ctx.Input<Tensor>("OutLinearBias");
    auto *out_linear_out = ctx.Output<Tensor>("OutLinearOut");

// bias+dropout+residual+layernorm
#if 1
    auto *ln_scale_2 = ctx.Input<Tensor>("Ln2Scale");
    auto *ln_bias_2 = ctx.Input<Tensor>("Ln2Bias");
    auto *dropout_mask_out = ctx.Output<Tensor>("DropoutMaskOut");
    auto *bias_dropout_residual_out =
        ctx.Output<Tensor>("BiasDropoutResidualOut");
    auto *ln_mean_2 = ctx.Output<Tensor>("Ln2Mean");
    auto *ln_var_2 = ctx.Output<Tensor>("Ln2Variance");
    const float ln2epsilon = ctx.Attr<float>("ln2epsilon");
#endif

#if 1
    float attn_dropout_prob = ctx.Attr<float>("attn_dropout_prob");
    std::cout << "limin: attn_dropout_prob = " << attn_dropout_prob
              << std::endl;
    bool is_test_1 = ctx.Attr<bool>("is_test1");
    auto &dropout_implementation_1 =
        ctx.Attr<std::string>("dropout_implementation1");
    bool is_upscale_in_train_1 =
        (dropout_implementation_1 == "upscale_in_train");
    auto *seed_1 = ctx.HasInput("Seed1") ? ctx.Input<Tensor>("Seed1") : nullptr;
    bool is_fix_seed_1 = ctx.Attr<bool>("fix_seed1");
    int seed_val_1 = ctx.Attr<int>("seed1");
#endif

    // final output.
    auto *out = ctx.Output<Tensor>("Y");

    // get data ptr for qkv part.
    const auto input_x_dims = input_x->dims();
    const auto qkv_w_dims = qkv_weight->dims();

    auto *x_data = input_x->data<T>();
    auto *ln_scale_data = (ln_scale == nullptr ? nullptr : ln_scale->data<U>());
    auto *ln_bias_data = (ln_bias == nullptr ? nullptr : ln_bias->data<U>());
    auto *ln_mean_data = ln_mean->mutable_data<U>(ctx.GetPlace());
    auto *ln_var_data = ln_var->mutable_data<U>(ctx.GetPlace());
    auto *ln_out_data = ln_out->mutable_data<T>(ctx.GetPlace());

    auto *qkv_weight_data = qkv_weight->data<T>();
    auto *qkv_bias_data = qkv_bias->data<T>();
    auto *qkv_out_data = qkv_out->mutable_data<T>(ctx.GetPlace());
    auto *qkv_bias_out_data = qkv_bias_out->mutable_data<T>(ctx.GetPlace());

    // get data ptr for FMHA.
    auto *src_mask_data = (src_mask == nullptr ? nullptr : src_mask->data<T>());
    auto *transpose_out_2_data =
        transpose_out_2->mutable_data<T>(ctx.GetPlace());
    auto *qk_out_data = qk_out->mutable_data<T>(ctx.GetPlace());
    auto *qktv_out_data = qktv_out->mutable_data<T>(ctx.GetPlace());
    auto *src_mask_out_data = src_mask_out->mutable_data<T>(ctx.GetPlace());
    auto *softmax_out_data = softmax_out->mutable_data<T>(ctx.GetPlace());
    auto *attn_dropout_mask_out_data =
        attn_dropout_mask_out->mutable_data<uint8_t>(ctx.GetPlace());
    auto *attn_dropout_out_data =
        attn_dropout_out->mutable_data<T>(ctx.GetPlace());
    auto *fmha_out_data = fmha_out->mutable_data<T>(ctx.GetPlace());

    // get data ptr for out_linear.
    auto *out_linear_weight_data = out_linear_weight->data<T>();
    auto *out_linear_bias_data = out_linear_bias->data<T>();
    auto *out_linear_out_data = out_linear_out->mutable_data<T>(ctx.GetPlace());

// get data ptr for bias+dropout+residual+layernorm
#if 1
    auto *ln_scale_2_data =
        (ln_scale_2 == nullptr ? nullptr : ln_scale_2->data<U>());
    auto *ln_bias_2_data =
        (ln_bias_2 == nullptr ? nullptr : ln_bias_2->data<U>());
    auto *dropout_mask_out_data =
        dropout_mask_out->mutable_data<uint8_t>(ctx.GetPlace());
    auto *bias_dropout_residual_out_data =
        bias_dropout_residual_out->mutable_data<T>(ctx.GetPlace());
    auto *ln_mean_2_data = ln_mean_2->mutable_data<U>(ctx.GetPlace());
    auto *ln_var_2_data = ln_var_2->mutable_data<U>(ctx.GetPlace());
#endif
    auto *final_out_data = out->mutable_data<T>(ctx.GetPlace());

    int batch_size = input_x_dims[0];
    int max_seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];

    int num_head = qkv_w_dims[1];
    int dim_head = qkv_w_dims[2];

    int bsz_seq = batch_size * max_seq_len;
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    bool transA = false;
    bool transB = true;
    bool compute_bias = true;
    auto layer_norm_compute = AttnLayerNorm<T>(ctx.cuda_device_context(),
                                               epsilon, bsz_seq, dim_embed);
    auto qkv_compute =
        AttnMatMul<T>(ctx.cuda_device_context(), transA, transB, bsz_seq,
                      output_size, input_size, compute_bias);

    // AttnDropoutParam(bool is_test, const std::string dropout_implementation,
    // float dropout_prob, bool is_upscale_in_train,
    // bool is_fix_seed, int seed_val, const Tensor* seed) {
    AttnDropoutParam attn_dropout_param(
        is_test_1, dropout_implementation_1, attn_dropout_prob,
        is_upscale_in_train_1, is_fix_seed_1, seed_val_1, seed_1);
    auto fmha_ref_compute =
        FMHARef<T>(ctx.cuda_device_context(), batch_size, max_seq_len, num_head,
                   dim_head, attn_dropout_param);
    // out_linear
    output_size = hidden_size;
    transA = false;
    transB = false;
    compute_bias = false;
    auto out_linear_compute =
        AttnMatMul<T>(ctx.cuda_device_context(), transA, transB, bsz_seq,
                      output_size, input_size, compute_bias);
#if 1
    DropoutParam dropout_param2(ctx, 0);
    FusedDropoutLayerNormHelper<T, uint8_t> fused_dropout_layernorm_helper(
        ctx.cuda_device_context(), bsz_seq, dim_embed, dropout_param2,
        ln2epsilon);
#endif

    // compute
    if (pre_layer_norm) {
      layer_norm_compute.ComputeForward(x_data, ln_scale_data, ln_bias_data,
                                        ln_out_data, ln_mean_data, ln_var_data);
      qkv_compute.ComputeForward(qkv_weight_data, ln_out_data, qkv_bias_data,
                                 qkv_out_data, qkv_bias_out_data);
    } else {
      qkv_compute.ComputeForward(qkv_weight_data, x_data, qkv_bias_data,
                                 qkv_out_data, qkv_bias_out_data);
    }
    // compute FMHA
    fmha_ref_compute.ComputeForward(*qkv_bias_out, *src_mask, transpose_out_2,
                                    qk_out, src_mask_out, softmax_out,
                                    attn_dropout_mask_out, attn_dropout_out,
                                    qktv_out, fmha_out);
// fmha_out: [batch_size, seq_len, num_head, head_dim]
// weight: [1024, 1024], [embed_dim, embed_dim]
// out_linear_out: [batch_size, seq_len, embed_dim]
#if 1
    out_linear_compute.ComputeForward(out_linear_weight_data, fmha_out_data,
                                      nullptr, out_linear_out_data, nullptr);
#endif
#if 1
    // out = layernorm(residual + dropout(src + bias))
    fused_dropout_layernorm_helper.LayernormResidualDropoutBias(
        ctx.cuda_device_context(), out_linear_out_data, x_data,
        out_linear_bias_data, ln_scale_2_data, ln_bias_2_data,
        bias_dropout_residual_out_data, dropout_mask_out_data, final_out_data,
        ln_mean_2_data, ln_var_2_data);
#endif
  }
};

template <typename T>
class FusedAttentionGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    const float epsilon = ctx.Attr<float>("epsilon");
#if 1
    const float ln2epsilon = ctx.Attr<float>("ln2epsilon");
#endif

#if 1
    float attn_dropout_prob = ctx.Attr<float>("attn_dropout_prob");
    bool is_test_1 = ctx.Attr<bool>("is_test1");
    auto &dropout_implementation_1 =
        ctx.Attr<std::string>("dropout_implementation1");
    bool is_upscale_in_train_1 =
        (dropout_implementation_1 == "upscale_in_train");
    auto *seed_1 = ctx.HasInput("Seed1") ? ctx.Input<Tensor>("Seed1") : nullptr;
    bool is_fix_seed_1 = ctx.Attr<bool>("fix_seed1");
    int seed_val_1 = ctx.Attr<int>("seed1");
#endif

    // get inputs.
    auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto *d_y_data = d_y->data<T>();

    // fw input
    auto *input_x = ctx.Input<Tensor>("X");
    auto *ln_scale = ctx.Input<Tensor>("LnScale");
#if 1
    auto *ln_2_scale = ctx.Input<Tensor>("Ln2Scale");
#endif
    auto *x_data = input_x->data<T>();
    auto *ln_scale_data = (ln_scale == nullptr ? nullptr : ln_scale->data<U>());
#if 1
    auto *ln_2_scale_data =
        (ln_2_scale == nullptr ? nullptr : ln_2_scale->data<U>());
#endif
    // fw parameters.
    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto *qkv_weight = ctx.Input<Tensor>("QKVW");
    auto *qkv_bias = ctx.Input<Tensor>("QKVBias");
    auto *out_linear_weight = ctx.Input<Tensor>("OutLinearW");
    auto *out_linear_bias = ctx.Input<Tensor>("OutLinearBias");
    auto *src_mask_data = (src_mask == nullptr ? nullptr : src_mask->data<T>());
    auto *qkv_weight_data = qkv_weight->data<T>();
    auto *qkv_bias_data = qkv_bias->data<T>();
    auto *out_linear_weight_data = out_linear_weight->data<T>();
    auto *out_linear_bias_data = out_linear_bias->data<T>();

    // fw output
    auto *ln_mean = ctx.Input<Tensor>("LnMean");
    auto *ln_var = ctx.Input<Tensor>("LnVariance");
    auto *ln_out = ctx.Input<Tensor>("LnOut");
    auto *fmha_out = ctx.Input<Tensor>("FMHAOut");
    auto *transpose_out_2 = ctx.Input<Tensor>("TransposeOut2");
    auto *qk_out = ctx.Input<Tensor>("QKOut");
    auto *qktv_out = ctx.Input<Tensor>("QKTVOut");
    auto *softmax_out = ctx.Input<Tensor>("SoftmaxOut");
    auto *attn_dropout_mask_out = ctx.Input<Tensor>("AttnDropoutMaskOut");
    auto *attn_dropout_out = ctx.Input<Tensor>("AttnDropoutOut");
    auto *src_mask_out = ctx.Input<Tensor>("SrcMaskOut");
    auto *out_linear_out = ctx.Input<Tensor>("OutLinearOut");
#if 1
    auto *ln_2_mean = ctx.Input<Tensor>("Ln2Mean");
    auto *ln_2_var = ctx.Input<Tensor>("Ln2Variance");
    auto *dropout_mask_out = ctx.Input<Tensor>("DropoutMaskOut");
    auto *bias_dropout_residual_out =
        ctx.Input<Tensor>("BiasDropoutResidualOut");
#endif
    auto *ln_mean_data = ln_mean->data<U>();
    auto *ln_var_data = ln_var->data<U>();
    auto *ln_out_data = ln_out->data<T>();
    auto *fmha_out_data = fmha_out->data<T>();
    auto *transpose_out_2_data = transpose_out_2->data<T>();
    auto *qk_out_data = qk_out->data<T>();
    auto *qktv_out_data = qktv_out->data<T>();
    auto *softmax_out_data = softmax_out->data<T>();
    auto *src_mask_out_data = src_mask_out->data<T>();
    auto *out_linear_out_data = out_linear_out->data<T>();
#if 1
    auto *ln_2_mean_data = ln_2_mean->data<U>();
    auto *ln_2_var_data = ln_2_var->data<U>();
    auto *dropout_mask_out_data = dropout_mask_out->data<uint8_t>();
    auto *bias_dropout_residual_out_data = bias_dropout_residual_out->data<T>();
#endif

    // bw output's grad
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_ln_out = ctx.Output<Tensor>(framework::GradVarName("LnOut"));
    auto *d_qkv_out = ctx.Output<Tensor>(framework::GradVarName("QKVOut"));
    auto *d_qkv_bias_out =
        ctx.Output<Tensor>(framework::GradVarName("QKVBiasOut"));
    auto *d_qktv_out = ctx.Output<Tensor>(framework::GradVarName("QKTVOut"));
    auto *d_transpose_out_2 =
        ctx.Output<Tensor>(framework::GradVarName("TransposeOut2"));
    auto *d_qk_out = ctx.Output<Tensor>(framework::GradVarName("QKOut"));
    auto *d_softmax_out =
        ctx.Output<Tensor>(framework::GradVarName("SoftmaxOut"));
    auto *d_attn_dropout_out =
        ctx.Output<Tensor>(framework::GradVarName("AttnDropoutOut"));
    auto *d_src_mask_out =
        ctx.Output<Tensor>(framework::GradVarName("SrcMaskOut"));
    auto *d_fmha_out = ctx.Output<Tensor>(framework::GradVarName("FMHAOut"));
    auto *d_out_linear_out =
        ctx.Output<Tensor>(framework::GradVarName("OutLinearOut"));
#if 1
    // auto *d_dropout_mask_out =
    //     ctx.Output<Tensor>(framework::GradVarName("DropoutMaskOut"));
    auto *d_bias_dropout_residual_out =
        ctx.Output<Tensor>(framework::GradVarName("BiasDropoutResidualOut"));
#endif
    auto *d_x_data = d_x->mutable_data<T>(ctx.GetPlace());
    auto *d_ln_out_data = d_ln_out->mutable_data<T>(ctx.GetPlace());
    auto *d_qkv_out_data = d_qkv_out->mutable_data<T>(ctx.GetPlace());
    auto *d_qkv_bias_out_data = d_qkv_bias_out->mutable_data<T>(ctx.GetPlace());
    auto *d_qktv_out_data = d_qktv_out->mutable_data<T>(ctx.GetPlace());
    auto *d_transpose_out_2_data =
        d_transpose_out_2->mutable_data<T>(ctx.GetPlace());
    auto *d_qk_out_data = d_qk_out->mutable_data<T>(ctx.GetPlace());
    auto *d_softmax_out_data = d_softmax_out->mutable_data<T>(ctx.GetPlace());
    auto *d_attn_dropout_out_data =
        d_attn_dropout_out->mutable_data<T>(ctx.GetPlace());
    auto *d_src_mask_out_data = d_src_mask_out->mutable_data<T>(ctx.GetPlace());
    auto *d_fmha_out_data = d_fmha_out->mutable_data<T>(ctx.GetPlace());
    auto *d_out_linear_out_data =
        d_out_linear_out->mutable_data<T>(ctx.GetPlace());
#if 1
    // auto *d_dropout_mask_out_data =
    //     d_dropout_mask_out->mutable_data<uint8_t>(ctx.GetPlace());
    auto *d_bias_dropout_residual_out_data =
        d_bias_dropout_residual_out->mutable_data<T>(ctx.GetPlace());
#endif

    // bw parameter's grad
    auto *d_ln_scale = ctx.Output<Tensor>(framework::GradVarName("LnScale"));
    auto *d_ln_bias = ctx.Output<Tensor>(framework::GradVarName("LnBias"));
    auto *d_qkv_weight = ctx.Output<Tensor>(framework::GradVarName("QKVW"));
    auto *d_qkv_bias = ctx.Output<Tensor>(framework::GradVarName("QKVBias"));
    auto *d_out_linear_weight =
        ctx.Output<Tensor>(framework::GradVarName("OutLinearW"));
    auto *d_out_linear_bias =
        ctx.Output<Tensor>(framework::GradVarName("OutLinearBias"));
#if 1
    auto *d_ln_2_scale = ctx.Output<Tensor>(framework::GradVarName("Ln2Scale"));
    auto *d_ln_2_bias = ctx.Output<Tensor>(framework::GradVarName("Ln2Bias"));
#endif
    auto *d_ln_scale_data =
        (d_ln_scale == nullptr ? nullptr
                               : d_ln_scale->mutable_data<U>(ctx.GetPlace()));
    auto *d_ln_bias_data =
        (d_ln_bias == nullptr ? nullptr
                              : d_ln_bias->mutable_data<U>(ctx.GetPlace()));
    auto *d_qkv_weight_data = d_qkv_weight->mutable_data<T>(ctx.GetPlace());
    auto *d_qkv_bias_data = d_qkv_bias->mutable_data<T>(ctx.GetPlace());
    auto *d_out_linear_weight_data =
        d_out_linear_weight->mutable_data<T>(ctx.GetPlace());
    auto *d_out_linear_bias_data =
        d_out_linear_bias->mutable_data<T>(ctx.GetPlace());
#if 1
    auto *d_ln_2_scale_data =
        (d_ln_2_scale == nullptr ? nullptr : d_ln_2_scale->mutable_data<U>(
                                                 ctx.GetPlace()));
    auto *d_ln_2_bias_data =
        (d_ln_2_bias == nullptr ? nullptr
                                : d_ln_2_bias->mutable_data<U>(ctx.GetPlace()));
#endif

    // get data ptr for qkv part.
    const auto input_x_dims = input_x->dims();
    const auto qkv_w_dims = qkv_weight->dims();

    int batch_size = input_x_dims[0];
    int max_seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];
    int num_head = qkv_w_dims[1];
    int dim_head = qkv_w_dims[2];

    int bsz_seq = batch_size * max_seq_len;
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    Tensor d_residual;
    d_residual.Resize(input_x_dims);
    T *d_residual_data = d_residual.mutable_data<T>(ctx.GetPlace());

    bool transA = false;
    bool transB = true;
    bool compute_bias = true;
    auto layer_norm_compute = AttnLayerNorm<T>(ctx.cuda_device_context(),
                                               epsilon, bsz_seq, dim_embed);
    auto qkv_compute =
        AttnMatMul<T>(ctx.cuda_device_context(), transA, transB, bsz_seq,
                      output_size, input_size, compute_bias);
    // fmha
    AttnDropoutParam attn_dropout_param(
        is_test_1, dropout_implementation_1, attn_dropout_prob,
        is_upscale_in_train_1, is_fix_seed_1, seed_val_1, seed_1);
    auto fmha_ref_compute =
        FMHARef<T>(ctx.cuda_device_context(), batch_size, max_seq_len, num_head,
                   dim_head, attn_dropout_param);
    // out_linear
    output_size = hidden_size;
    transA = false;
    transB = false;
    compute_bias = false;
    auto out_linear_compute =
        AttnMatMul<T>(ctx.cuda_device_context(), transA, transB, bsz_seq,
                      output_size, input_size, compute_bias);
#if 1
    // bias + dropout + residual + layernorm.
    DropoutParam dropout_param2(ctx, 0);
    FusedDropoutLayerNormHelper<T, uint8_t> fused_dropout_layernorm_helper(
        ctx.cuda_device_context(), bsz_seq, dim_embed, dropout_param2,
        ln2epsilon);
#endif
#if 1
    // dout -> dlayernorm_dsrc, dscale, layernorm_dbias
    // dlayernorm_dsrc -> dsrc, dbias, dresidual
    fused_dropout_layernorm_helper.LayernormResidualDropoutBiasGrad(
        ctx.cuda_device_context(), d_y_data, bias_dropout_residual_out_data,
        dropout_mask_out_data, ln_2_scale_data, ln_2_mean_data, ln_2_var_data,
        d_bias_dropout_residual_out_data, d_ln_2_scale_data, d_ln_2_bias_data,
        d_out_linear_out_data, d_out_linear_bias_data, d_residual_data);
#endif
#if 1
    out_linear_compute.ComputeBackward(fmha_out_data, out_linear_weight_data,
                                       d_out_linear_out_data, d_fmha_out_data,
                                       d_out_linear_weight_data, nullptr);
#endif
#if 1
    fmha_ref_compute.ComputeBackward(
        *transpose_out_2, *src_mask, *softmax_out, *attn_dropout_mask_out,
        *attn_dropout_out, *qk_out, *src_mask_out, *d_fmha_out, d_qktv_out,
        d_attn_dropout_out, d_softmax_out, d_src_mask_out, d_qk_out,
        d_transpose_out_2, nullptr, d_qkv_bias_out);
    // d_qkv_bias_out->d_qkv_out
    // batch_size, seq_len, 3, num_head, head_size
    hipMemcpyAsync(d_qkv_out_data, d_qkv_bias_out_data,
                    bsz_seq * 3 * num_head * dim_head * sizeof(T),
                    hipMemcpyDeviceToDevice);
#endif
#if 1
    // get qkv
    if (pre_layer_norm) {
      qkv_compute.ComputeBackward(ln_out_data, qkv_weight_data,
                                  d_qkv_bias_out_data, d_ln_out_data,
                                  d_qkv_weight_data, d_qkv_bias_data);
      layer_norm_compute.ComputeBackward(x_data, d_ln_out_data, ln_scale_data,
                                         ln_mean_data, ln_var_data, d_x_data,
                                         d_ln_scale_data, d_ln_bias_data);
    } else {
      qkv_compute.ComputeBackward(x_data, qkv_weight_data, d_qkv_bias_out_data,
                                  d_x_data, d_qkv_weight_data, d_qkv_bias_data);
    }
    // gradient accumulation: d_x[] + d_residual[] = d_x[]
    std::vector<const Tensor *> ins;
    std::vector<Tensor *> outs;
    ins.emplace_back(&d_residual);
    ins.emplace_back(d_x);
    outs.emplace_back(d_x);
    int elewise_add_axis = -1;
    LaunchElementwiseCudaKernel<ElementwiseType::kBinary, T, T>(
        ctx.cuda_device_context(), ins, &outs, elewise_add_axis,
        AddFunctor<T>());
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(fused_attention, ops::FusedAttentionOpKernel<float>,
                        ops::FusedAttentionOpKernel<double>,
                        ops::FusedAttentionOpKernel<plat::float16>);
REGISTER_OP_CUDA_KERNEL(fused_attention_grad,
                        ops::FusedAttentionGradKernel<float>,
                        ops::FusedAttentionGradKernel<double>,
                        ops::FusedAttentionGradKernel<plat::float16>);
