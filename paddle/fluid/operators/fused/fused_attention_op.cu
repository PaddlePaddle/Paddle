/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/platform/cuda_device_function.h"

#ifdef PADDLE_WITH_CUDA
#include "paddle/fluid/platform/cudnn_helper.h"
#endif
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/platform/miopen_helper.h"
#endif

#include <hip/hip_fp16.h>
#include "paddle/fluid/operators/elementwise/elementwise_add_op.h"
#include "paddle/fluid/operators/math/math_function.h"

#include "paddle/fluid/operators/fused/fused_attention_op.h"

#include "paddle/fluid/operators/fused/attention_layer_norm.h"
#include "paddle/fluid/operators/fused/attn_gemm.h"
#include "paddle/fluid/operators/fused/fmha_ref.h"
#include "paddle/fluid/operators/fused/fused_dropout_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
class FusedAttentionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    using U = LayerNormParamType<T>;
    auto *input_x = ctx.Input<Tensor>("X");

    const auto pre_layer_norm = ctx.Attr<bool>("pre_layer_norm");
    const float epsilon = ctx.Attr<float>("epsilon");
    auto *ln_scale = ctx.Input<Tensor>("LnScale");
    auto *ln_bias = ctx.Input<Tensor>("LnBias");
    auto *ln_mean = ctx.Output<Tensor>("LnMean");
    auto *ln_var = ctx.Output<Tensor>("LnVariance");
    auto *ln_out = ctx.Output<Tensor>("LnOut");

    // x: qkv's input [batch_size, seq_len, dim_embed]
    // y: qkv's weight: [3, num_head, dim_head, dim_embed]
    auto *qkv_weight = ctx.Input<Tensor>("QKVW");
    auto *qkv_bias = ctx.Input<Tensor>("QKVBias");
    auto *qkv_out = ctx.Output<Tensor>("QKVOut");
    auto *qkv_bias_out = ctx.Output<Tensor>("QKVBiasOut");

    auto *src_mask = ctx.Input<Tensor>("SrcMask");
    auto *transpose_out_2 = ctx.Output<Tensor>("TransposeOut2");
    auto *qk_out = ctx.Output<Tensor>("QKOut");
    auto *qktv_out = ctx.Output<Tensor>("QKTVOut");
    auto *softmax_out = ctx.Output<Tensor>("SoftmaxOut");
    auto *attn_dropout_mask_out = ctx.Output<Tensor>("AttnDropoutMaskOut");
    auto *attn_dropout_out = ctx.Output<Tensor>("AttnDropoutOut");
    auto *src_mask_out = ctx.Output<Tensor>("SrcMaskOut");
    auto *fmha_out = ctx.Output<Tensor>("FMHAOut");

    auto *out_linear_weight = ctx.Input<Tensor>("OutLinearW");
    auto *out_linear_bias = ctx.Input<Tensor>("OutLinearBias");
    auto *out_linear_out = ctx.Output<Tensor>("OutLinearOut");

    auto *ln_scale_2 = ctx.Input<Tensor>("Ln2Scale");
    auto *ln_bias_2 = ctx.Input<Tensor>("Ln2Bias");
    auto *dropout_mask_out = ctx.Output<Tensor>("DropoutMaskOut");
    auto *bias_dropout_residual_out =
        ctx.Output<Tensor>("BiasDropoutResidualOut");
    auto *ln_mean_2 = ctx.Output<Tensor>("Ln2Mean");
    auto *ln_var_2 = ctx.Output<Tensor>("Ln2Variance");
    const float ln2epsilon = ctx.Attr<float>("ln2epsilon");

    float attn_dropout_prob = ctx.Attr<float>("attn_dropout_prob");
    bool is_test_1 = ctx.Attr<bool>("is_test1");
    auto &dropout_implementation_1 =
        ctx.Attr<std::string>("dropout_implementation1");
    bool is_upscale_in_train_1 =
        (dropout_implementation_1 == "upscale_in_train");
    auto *seed_1 = ctx.HasInput("Seed1") ? ctx.Input<Tensor>("Seed1") : nullptr;
    bool is_fix_seed_1 = ctx.Attr<bool>("fix_seed1");
    int seed_val_1 = ctx.Attr<int>("seed1");

    // final output.
    auto *out = ctx.Output<Tensor>("Y");

    // get data ptr for qkv part.
    const auto input_x_dims = input_x->dims();
    const auto qkv_w_dims = qkv_weight->dims();

    auto *x_data = input_x->data<T>();
    auto *ln_scale_data = (ln_scale == nullptr ? nullptr : ln_scale->data<U>());
    auto *ln_bias_data = (ln_bias == nullptr ? nullptr : ln_bias->data<U>());
    auto *ln_mean_data = ln_mean->mutable_data<U>(ctx.GetPlace());
    auto *ln_var_data = ln_var->mutable_data<U>(ctx.GetPlace());
    auto *ln_out_data = ln_out->mutable_data<T>(ctx.GetPlace());

    auto *qkv_weight_data = qkv_weight->data<T>();
    auto *qkv_bias_data = qkv_bias->data<T>();
    auto *qkv_out_data = qkv_out->mutable_data<T>(ctx.GetPlace());
    auto *qkv_bias_out_data = qkv_bias_out->mutable_data<T>(ctx.GetPlace());

    // get data ptr for FMHA.
    auto *src_mask_data = (src_mask == nullptr ? nullptr : src_mask->data<T>());
    auto *transpose_out_2_data =
        transpose_out_2->mutable_data<T>(ctx.GetPlace());
    auto *qk_out_data = qk_out->mutable_data<T>(ctx.GetPlace());
    auto *qktv_out_data = qktv_out->mutable_data<T>(ctx.GetPlace());
    auto *src_mask_out_data = src_mask_out->mutable_data<T>(ctx.GetPlace());
    auto *softmax_out_data = softmax_out->mutable_data<T>(ctx.GetPlace());
    auto *attn_dropout_mask_out_data =
        attn_dropout_mask_out->mutable_data<uint8_t>(ctx.GetPlace());
    auto *attn_dropout_out_data =
        attn_dropout_out->mutable_data<T>(ctx.GetPlace());
    auto *fmha_out_data = fmha_out->mutable_data<T>(ctx.GetPlace());

    // get data ptr for out_linear.
    auto *out_linear_weight_data = out_linear_weight->data<T>();
    auto *out_linear_bias_data = out_linear_bias->data<T>();
    auto *out_linear_out_data = out_linear_out->mutable_data<T>(ctx.GetPlace());

    // get data ptr for bias+dropout+residual+layernorm
    auto *ln_scale_2_data =
        (ln_scale_2 == nullptr ? nullptr : ln_scale_2->data<U>());
    auto *ln_bias_2_data =
        (ln_bias_2 == nullptr ? nullptr : ln_bias_2->data<U>());
    auto *dropout_mask_out_data =
        dropout_mask_out->mutable_data<uint8_t>(ctx.GetPlace());
    auto *bias_dropout_residual_out_data =
        bias_dropout_residual_out->mutable_data<T>(ctx.GetPlace());
    auto *ln_mean_2_data = ln_mean_2->mutable_data<U>(ctx.GetPlace());
    auto *ln_var_2_data = ln_var_2->mutable_data<U>(ctx.GetPlace());
    auto *final_out_data = out->mutable_data<T>(ctx.GetPlace());

    int batch_size = input_x_dims[0];
    int max_seq_len = input_x_dims[1];
    int dim_embed = input_x_dims[2];

    int num_head = qkv_w_dims[1];
    int dim_head = qkv_w_dims[2];

    int bsz_seq = batch_size * max_seq_len;
    int hidden_size = num_head * dim_head;
    int output_size = 3 * hidden_size;
    int input_size = dim_embed;

    bool transA = false;
    bool transB = true;
    bool compute_bias = true;
    auto layer_norm_compute = AttnLayerNorm<T>(ctx.cuda_device_context(),
                                               epsilon, bsz_seq, dim_embed);
    auto qkv_compute =
        AttnMatMul<T>(ctx.cuda_device_context(), transA, transB, bsz_seq,
                      output_size, input_size, compute_bias);

    AttnDropoutParam attn_dropout_param(
        is_test_1, dropout_implementation_1, attn_dropout_prob,
        is_upscale_in_train_1, is_fix_seed_1, seed_val_1, seed_1);
    auto fmha_ref_compute =
        FMHARef<T>(ctx.cuda_device_context(), batch_size, max_seq_len, num_head,
                   dim_head, attn_dropout_param);

    output_size = hidden_size;
    transA = false;
    transB = false;
    compute_bias = false;
    auto out_linear_compute =
        AttnMatMul<T>(ctx.cuda_device_context(), transA, transB, bsz_seq,
                      output_size, input_size, compute_bias);
    DropoutParam dropout_param2(ctx, 0);
    FusedDropoutLayerNormHelper<T, uint8_t> fused_dropout_layernorm_helper(
        ctx.cuda_device_context(), bsz_seq, dim_embed, dropout_param2,
        ln2epsilon);

    if (pre_layer_norm) {
      layer_norm_compute.ComputeForward(x_data, ln_scale_data, ln_bias_data,
                                        ln_out_data, ln_mean_data, ln_var_data);
      qkv_compute.ComputeForward(qkv_weight_data, ln_out_data, qkv_bias_data,
                                 qkv_out_data, qkv_bias_out_data);
    } else {
      qkv_compute.ComputeForward(qkv_weight_data, x_data, qkv_bias_data,
                                 qkv_out_data, qkv_bias_out_data);
    }
    fmha_ref_compute.ComputeForward(*qkv_bias_out, *src_mask, transpose_out_2,
                                    qk_out, src_mask_out, softmax_out,
                                    attn_dropout_mask_out, attn_dropout_out,
                                    qktv_out, fmha_out);
    // fmha_out: [batch_size, seq_len, num_head, head_dim]
    // weight: [1024, 1024], [embed_dim, embed_dim]
    // out_linear_out: [batch_size, seq_len, embed_dim]
    out_linear_compute.ComputeForward(out_linear_weight_data, fmha_out_data,
                                      nullptr, out_linear_out_data, nullptr);
    // out = layernorm(residual + dropout(src + bias))
    fused_dropout_layernorm_helper.LayernormResidualDropoutBias(
        ctx.cuda_device_context(), out_linear_out_data, x_data,
        out_linear_bias_data, ln_scale_2_data, ln_bias_2_data,
        bias_dropout_residual_out_data, dropout_mask_out_data, final_out_data,
        ln_mean_2_data, ln_var_2_data);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(fused_attention, ops::FusedAttentionOpKernel<float>,
                        ops::FusedAttentionOpKernel<double>,
                        ops::FusedAttentionOpKernel<plat::float16>);
