#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <paddle/fluid/platform/device_context.h>
#include <algorithm>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/detail/safe_ref.h"
#include "paddle/fluid/operators/math/blas.h"

namespace paddle {
namespace operators {

#define FINAL_MASK 0xffffffff
#define HALF_WARP 16
#define WARP_SIZE 32

template <typename T>
__inline__ __device__ T warpReduceSum(T val, unsigned lane_mask) {
  for (int mask = HALF_WARP; mask > 0; mask >>= 1)
#if __CUDA_ARCH__ >= 350 && CUDA_VERSION >= 9000
    val += __shfl_xor_sync(lane_mask, val, mask, warpSize);
#else
    val += __shfl_xor(val, mask, warpSize);
#endif
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
__inline__ __device__ T blockReduceSum(T val, unsigned mask) {
  static __shared__ T shared[WARP_SIZE];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  val = warpReduceSum<T>(val, mask);

  if (lane == 0) shared[wid] = val;

  __syncthreads();

  // align block_span to warpSize
  int block_span = (blockDim.x + warpSize - 1) >> 5;
  val = (threadIdx.x < block_span) ? shared[lane] : static_cast<T>(0.0f);
  val = warpReduceSum<T>(val, mask);

  return val;
}

template <typename T>
__inline__ __device__ T warpReduceMax(T val, unsigned lane_mask) {
  for (int mask = HALF_WARP; mask > 0; mask >>= 1)
#if __CUDA_ARCH__ >= 350 && CUDA_VERSION >= 9000
    val = max(val, __shfl_xor_sync(lane_mask, val, mask, warpSize));
#else
    val = max(val, __shfl_xor(val, mask, warpSize));
#endif
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
__inline__ __device__ T blockReduceMax(T val, unsigned mask) {
  static __shared__ T shared[WARP_SIZE];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  val = warpReduceMax(val, mask);

  if (lane == 0) shared[wid] = val;

  __syncthreads();

  // align block_span to warpSize
  int block_span = (blockDim.x + warpSize - 1) >> 5;
  val = (threadIdx.x < block_span) ? shared[lane] : -1e10f;
  val = warpReduceMax(val, mask);

  return val;
}

template <typename T>
__global__ void add_QKV(const T *Q, const T *K, const T *V, T *q_buf_,
                        T *k_buf_, T *v_buf_, const T *bias_q, const T *bias_k,
                        const T *bias_v, int batch_size, int seq_len,
                        int head_num, int size_per_head) {
  const T *data_ptr_q, *data_ptr_k, *data_ptr_v;
  const T *bias_ptr_q, *bias_ptr_k, *bias_ptr_v;

  int m = batch_size * seq_len;
  int n = head_num * size_per_head;

  int row_offset = (blockIdx.x % m) * n;

  data_ptr_q = Q + row_offset;
  data_ptr_k = K + row_offset;
  data_ptr_v = V + row_offset;
  // bias ptr
  bias_ptr_q = bias_q;
  bias_ptr_k = bias_k;
  bias_ptr_v = bias_v;

  int batch_id = (blockIdx.x % m) / seq_len;
  int head_id = threadIdx.x / size_per_head;
  int id_in_head = threadIdx.x % size_per_head;
  int word_start_id = (blockIdx.x) % seq_len;

#if __CUDA_ARCH__ >= 350
  T tmp_q = __ldg(&data_ptr_q[threadIdx.x]) + __ldg(&bias_ptr_q[threadIdx.x]);
  T tmp_k = __ldg(&data_ptr_k[threadIdx.x]) + __ldg(&bias_ptr_k[threadIdx.x]);
  T tmp_v = __ldg(&data_ptr_v[threadIdx.x]) + __ldg(&bias_ptr_v[threadIdx.x]);
#else
  T tmp_q = data_ptr_q[threadIdx.x] + bias_ptr_q[threadIdx.x];
  T tmp_k = data_ptr_k[threadIdx.x] + bias_ptr_k[threadIdx.x];
  T tmp_v = data_ptr_v[threadIdx.x] + bias_ptr_v[threadIdx.x];
#endif

  int target_id = batch_id * (seq_len * head_num * size_per_head) +
                  head_id * seq_len * size_per_head +
                  word_start_id * size_per_head + id_in_head;

  q_buf_[target_id] = tmp_q;
  k_buf_[target_id] = tmp_k;
  v_buf_[target_id] = tmp_v;
}

// Keep to compare performance
template <typename T>
__global__ void add_QKV_V2(const T *Q, const T *K, const T *V, T *q_buf_,
                           T *k_buf_, T *v_buf_, const T *bias_Q,
                           const T *bias_K, const T *bias_V, int batch_size,
                           int seq_len, int head_num, int size_per_head,
                           const int word_per_block) {
  const T *data_ptr;
  T *buf_ptr;
  const T *bias_ptr;

  int m = batch_size * seq_len;
  int n = head_num * size_per_head;

  int qkv_id = blockIdx.x * word_per_block / m;
  int row_offset = (blockIdx.x * word_per_block % m) * n;

  if (qkv_id == 0) {
    data_ptr = Q + row_offset;
    buf_ptr = q_buf_;
    bias_ptr = bias_Q;
  } else if (qkv_id == 1) {
    data_ptr = K + row_offset;
    buf_ptr = k_buf_;
    bias_ptr = bias_K;
  } else {
    data_ptr = V + row_offset;
    buf_ptr = v_buf_;
    bias_ptr = bias_V;
  }

  int batch_id = (blockIdx.x * word_per_block % m) / seq_len;
  int head_id = threadIdx.x / size_per_head;
  int id_in_head = threadIdx.x % size_per_head;
  int word_start_id = (blockIdx.x * word_per_block) % seq_len;

#if __CUDA_ARCH__ >= 350
  T bias = __ldg(&bias_ptr[threadIdx.x]);
#else
  T bias = bias_ptr[threadIdx.x];
#endif

  for (int i = word_start_id; i < word_start_id + word_per_block; ++i) {
    T tmp = data_ptr[threadIdx.x] + bias;

    int target_id = batch_id * (seq_len * head_num * size_per_head) +
                    head_id * seq_len * size_per_head + i * size_per_head +
                    id_in_head;

    buf_ptr[target_id] = tmp;
    data_ptr += n;
  }
}

template <typename T>
__global__ void softmax_kernel_with_eltadd(T *qk_buf_, const T *bias_qk_,
                                           const int batch_size,
                                           const int head_num,
                                           const int seq_len,
                                           const unsigned mask) {
  int qk_offset = blockIdx.x * seq_len;
  assert(blockDim.x % 32 == 0);

  __shared__ float s_sum, s_max;

  float qk = threadIdx.x < seq_len
                 ? static_cast<float>((qk_buf_[threadIdx.x + qk_offset] +
                                       bias_qk_[threadIdx.x + qk_offset]))
                 : 0.0f;
  float tmp = threadIdx.x < seq_len ? static_cast<float>(qk) : -1e20f;

  float max_val = blockReduceMax<float>(tmp, mask);

  if (threadIdx.x == 0) s_max = max_val;
  __syncthreads();

  float qk_tmp =
      threadIdx.x < seq_len ? __expf(static_cast<float>(tmp - s_max)) : 0.0f;
  float sum_val = blockReduceSum<float>(qk_tmp, mask);

  if (threadIdx.x == 0) {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

// For verify result
template <typename T>
__global__ void elt_qk_add(const T *bias_qk, T *qk_buf, int head_num,
                           int seq_len, int size_per_head, int batch_size) {
  int m = batch_size * head_num * seq_len;
  int row_id = blockIdx.x % m;
  int dst_id = row_id * seq_len + threadIdx.x;
  const T *bias_ptr = bias_qk;
#if __CUDA_ARCH__ >= 350
  int tmp_bias = __ldg(&bias_ptr[dst_id]);
#else
  int tmp_bias = bias_ptr[dst_id];
#endif

  qk_buf[dst_id] += tmp_bias;
}

// Compute Q*K->softmax->eltadd
template <typename T>
void MatMulWithHeadQK(const platform::CUDADeviceContext &context, int head_num,
                      int seq_len, int size_per_head, int batch_size,
                      bool q_trans, bool k_trans, T *q_buf_, T *k_buf_,
                      T *qk_buf_, const T *bias_qk, T alpha, T beta) {
  CBLAS_TRANSPOSE transA = !q_trans ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = !k_trans ? CblasNoTrans : CblasTrans;

  auto blas = math::GetBlas<platform::CUDADeviceContext, T>(context);
  auto stream = context.stream();

  blas.BatchedGEMM(transA, transB, seq_len, seq_len, size_per_head, alpha,
                   q_buf_, k_buf_, beta, qk_buf_, batch_size * head_num,
                   seq_len * size_per_head, seq_len * size_per_head);

  int grid = batch_size * head_num * seq_len;
  int block = seq_len;

  // Align block to 32, also limit seq_len to max block size.
  PADDLE_ENFORCE_LE(seq_len, 1024, platform::errors::InvalidArgument(
                                       "seq_len should <= 1024, "
                                       "but received seq_len is:%d",
                                       seq_len));
  if (seq_len <= 32)
    block = 32;
  else if (seq_len > 32 && seq_len <= 64)
    block = 64;
  else if (seq_len > 64 && seq_len <= 128)
    block = 128;
  else if (seq_len > 128 && seq_len <= 256)
    block = 256;
  else if (seq_len > 256 && seq_len <= 512)
    block = 512;
  else
    block = 1024;

  softmax_kernel_with_eltadd<T><<<grid, block, 0, stream>>>(
      qk_buf_, bias_qk, batch_size, head_num, seq_len, FINAL_MASK);
}

template <typename T>
__global__ void transpose(T *src, T *dst, const int batch_size,
                          const int seq_len, const int head_num,
                          const int size_per_head) {
  int batch_id = blockIdx.x / (head_num * seq_len);
  int seq_id = blockIdx.x % seq_len;
  int head_id = (blockIdx.x % (head_num * seq_len)) / seq_len;
  dst[batch_id * (head_num * seq_len * size_per_head) +
      seq_id * head_num * size_per_head + head_id * size_per_head +
      threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

// Compute QK*V->transpose
template <typename T>
void MatMulWithHeadQKV(const platform::CUDADeviceContext &context, int head_num,
                       int seq_len, int size_per_head, int batch_size,
                       bool qk_trans, bool v_trans, T *v_buf_, const T *qk_buf_,
                       T *dst, T *out, T alpha, T beta) {
  int m = batch_size * seq_len;
  int k = head_num * size_per_head;

  auto blas = math::GetBlas<platform::CUDADeviceContext, T>(context);
  auto stream = context.stream();
  CBLAS_TRANSPOSE transA = !qk_trans ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = !v_trans ? CblasNoTrans : CblasTrans;

  blas.BatchedGEMM(transA, transB, seq_len, size_per_head, seq_len, alpha,
                   qk_buf_, v_buf_, beta, dst, batch_size * head_num,
                   seq_len * seq_len, seq_len * size_per_head);

  int grid = batch_size * head_num * seq_len;
  int block = size_per_head;
  transpose<T><<<grid, block, 0, stream>>>(dst, out, batch_size, seq_len,
                                           head_num, size_per_head);
}

template <typename T>
void MultiHeadGPUCompute(const platform::CUDADeviceContext &dev_ctx,
                         int head_num, const framework::DDim &mat_q,
                         const framework::DDim &mat_k,
                         const framework::DDim &mat_v, const T *Q, const T *K,
                         const T *V, const T *bias_q, const T *bias_k,
                         const T *bias_v, const T *bias_qk, T *out, T alpha,
                         T beta, bool trans_q, bool trans_k, bool trans_v) {
  int seq_len = mat_q[1];
  int size_per_head = (mat_q[2] / head_num);
  int batch_size = mat_q[0];
  int buf_size = batch_size * head_num * seq_len * size_per_head;
  int qk_buf_size = batch_size * head_num * seq_len * seq_len;

  auto alloc_buf =
      memory::Alloc(dev_ctx, (buf_size * 4 + qk_buf_size) * sizeof(T));

  T *buf = reinterpret_cast<T *>(alloc_buf->ptr());
  T *q_buf = buf;
  T *k_buf = buf + buf_size;
  T *v_buf = buf + 2 * buf_size;
  T *qk_buf = buf + 3 * buf_size;
  T *dst_buf = buf + 3 * buf_size + qk_buf_size;

  int m = batch_size * seq_len;
  int k = head_num * size_per_head;

  // Each block process head*size-per_head element,
  // have m lines. bias is m lines
  auto blas = math::GetBlas<platform::CUDADeviceContext, T>(dev_ctx);
  auto stream = dev_ctx.stream();

  int grid = m;
  PADDLE_ENFORCE_LE(k, 1024,
                    "Input head_number * size_per_head should <= 1024");
  int block = k <= 1024 ? k : 1024;
  add_QKV<T><<<grid, block, 0, stream>>>(Q, K, V, q_buf, k_buf, v_buf, bias_q,
                                         bias_k, bias_v, batch_size, seq_len,
                                         head_num, size_per_head);

  MatMulWithHeadQK<T>(dev_ctx, head_num, seq_len, size_per_head, batch_size,
                      trans_q, trans_k, q_buf, k_buf, qk_buf, bias_qk, alpha,
                      beta);
  MatMulWithHeadQKV<T>(dev_ctx, head_num, seq_len, size_per_head, batch_size,
                       false, trans_v, v_buf, qk_buf, dst_buf, out, T(1.0),
                       beta);
}

template <typename DeviceContext, typename T>
class MultiHeadMatMulKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *q = context.Input<framework::Tensor>("Q");
    auto *k = context.Input<framework::Tensor>("K");
    auto *v = context.Input<framework::Tensor>("V");

    auto &bias_q = detail::Ref(context.Input<framework::Tensor>("BiasQ"),
                               "Cannot find BiasQ");
    auto &bias_k = detail::Ref(context.Input<framework::Tensor>("BiasK"),
                               "Cannot find BiasK");
    auto &bias_v = detail::Ref(context.Input<framework::Tensor>("BiasV"),
                               "Cannot find BiasV");

    auto &bias_qk = detail::Ref(context.Input<framework::Tensor>("BiasQK"),
                                "Cannot find QK");

    auto *out = context.Output<framework::Tensor>("Out");
    out->mutable_data<T>(context.GetPlace());

    T scale = static_cast<T>(context.Attr<float>("alpha"));
    bool transpose_q = context.Attr<bool>("transpose_Q");
    bool transpose_k = context.Attr<bool>("transpose_K");
    bool transpose_v = context.Attr<bool>("transpose_V");

    int head_number = context.Attr<int>("head_number");
    // compute q*k with eltadd
    auto &device_ctx = context.template device_context<DeviceContext>();

    MultiHeadGPUCompute<T>(device_ctx, head_number, q->dims(), k->dims(),
                           v->dims(), q->data<T>(), k->data<T>(), v->data<T>(),
                           bias_q.data<T>(), bias_k.data<T>(), bias_v.data<T>(),
                           bias_qk.data<T>(), out->data<T>(), scale, T(0.0),
                           transpose_q, transpose_k, transpose_v);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    multihead_matmul,
    ops::MultiHeadMatMulKernel<paddle::platform::CUDADeviceContext, float>,
    ops::MultiHeadMatMulKernel<paddle::platform::CUDADeviceContext, double>);
