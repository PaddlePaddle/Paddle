#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <paddle/fluid/platform/device_context.h>
#include <algorithm>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"
#include "paddle/fluid/operators/math/blas.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void transpose(T *src, T *dst, const int batch_size,
                          const int seq_len, const int head_num,
                          const int size_per_head) {
  int batch_id = blockIdx.x / (head_num * seq_len);
  int seq_id = blockIdx.x % seq_len;
  int head_id = (blockIdx.x % (head_num * seq_len)) / seq_len;
  dst[batch_id * (head_num * seq_len * size_per_head) +
      seq_id * head_num * size_per_head + head_id * size_per_head +
      threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

template <typename T>
inline __device__ T add_func(T a, T b);

template <>
__device__ float add_func<float>(float a, float b) {
  return a + b;
}

template <>
__device__ float2 add_func<float2>(float2 a, float2 b) {
  float2 c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

template <>
__device__ float4 add_func<float4>(float4 a, float4 b) {
  float4 c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  c.z = a.z + b.z;
  c.w = a.w + b.w;
  return c;
}

template <typename T>
__global__ void TransposeQkvKernel(const int H, const T *input, const T *bias,
                                   T *output) {
  // Input: BxSx3xNxH
  // Bias: 3xSxB
  // Output: 3xBxNxSxH
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;

  const int N = blockDim.y;
  const int S = gridDim.x;
  const int B = gridDim.y;

  const int NH = N * H;
  const int NHS = NH * S;
  const int in_offset = n * H + m * NH + s * 3 * NH + b * NHS * 3;
  const int bias_offset = m * NH + n * H;
  const int out_offset = s * H + n * S * H + b * NHS + m * NHS * B;

  const int i = threadIdx.x;
  output[out_offset + i] =
      add_func(input[in_offset + i], bias[bias_offset + i]);
}

void TransQKVWithBias(const int batch, const int seq_len, const int head_size,
                      const int head_num, const float *input, const float *bias,
                      float *output, hipStream_t stream) {
  // BxSx3xNxH + 3xNxH -> 3xBxNxSxH
  int scratch_size = batch * head_num * seq_len * seq_len;
  const dim3 grid(seq_len, batch, 3);
  // scratch % 4 == 0 to ensure the alignment
  if (head_size % 4 == 0 && scratch_size % 4 == 0) {
    const int h = head_size / 4;
    const float4 *input4 = reinterpret_cast<const float4 *>(input);
    const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
    float4 *output4 = reinterpret_cast<float4 *>(output);
    const dim3 block(h, head_num, 1);

    // limit h * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(h * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024 * 4));
    TransposeQkvKernel<float4><<<grid, block, 0, stream>>>(h, input4, bias4,
                                                           output4);
  } else if (head_size % 2 == 0 && scratch_size % 2 == 0) {
    const int h = head_size / 2;
    const float2 *input2 = reinterpret_cast<const float2 *>(input);
    const float2 *bias2 = reinterpret_cast<const float2 *>(bias);
    float2 *output2 = reinterpret_cast<float2 *>(output);
    const dim3 block(h, head_num, 1);
    // limit h * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(h * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024 * 2));
    TransposeQkvKernel<float2><<<grid, block, 0, stream>>>(h, input2, bias2,
                                                           output2);
  } else {
    const dim3 block(head_size, head_num, 1);
    // limit head_size * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(head_size * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024));
    TransposeQkvKernel<float><<<grid, block, 0, stream>>>(head_size, input,
                                                          bias, output);
  }
}

template <typename DeviceContext, typename T>
class MultiHeadMatMulV2Kernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    using Tensor = framework::Tensor;
    auto *input = context.Input<framework::Tensor>("Input");
    auto *w = context.Input<framework::Tensor>("W");
    auto *bias = context.Input<framework::Tensor>("Bias");
    auto &bias_qk = GET_DATA_SAFELY(context.Input<framework::Tensor>("BiasQK"),
                                    "Input", "BiasQK", "MultiHeadMatMulV2");

    auto *input_d = input->data<T>();
    auto *w_d = w->data<T>();
    auto *bias_d = bias->data<T>();
    auto *bias_qk_d = bias_qk.template data<T>();
    T scale = static_cast<T>(context.Attr<float>("alpha"));

    int head_number = context.Attr<int>("head_number");
    // compute q*k with eltadd
    auto &device_ctx = context.template device_context<DeviceContext>();
    // should be (B * S * hidden)
    auto input_dims = input->dims();
    // shouble be (hidden * 3 * all_head_size)
    auto w_dims = w->dims();
    int batch = input_dims[0];
    int seq_len = input_dims[1];
    int hidden = input_dims[2];

    int all_head_size = w_dims[2];
    int head_size = all_head_size / head_number;

    auto *out = context.Output<framework::Tensor>("Out");
    out->Resize({batch, seq_len, all_head_size});
    auto *output_d = out->mutable_data<T>(context.GetPlace());

    // (B*S, hidden)
    const Tensor input_matrix =
        framework::ReshapeToMatrix(*input, 2 /*x_num_col_dims */);
    // (hidden, 3 * all_head_size)
    const Tensor w_matrix =
        framework::ReshapeToMatrix(*w, 1 /*y_num_col_dims*/);

    Tensor temp_out_tensor;
    auto temp_out_dims =
        framework::make_ddim({batch, seq_len, 3, head_number, head_size});
    temp_out_tensor.Resize({batch * seq_len, framework::product(temp_out_dims) /
                                                 (batch * seq_len)});
    auto *temp_out_data = temp_out_tensor.mutable_data<T>(context.GetPlace());

    // (B * S, hidden) * (hidden, 3 * N * H) -> (B * S * 3 * N * H)
    auto blas = math::GetBlas<platform::CUDADeviceContext, T>(device_ctx);
    blas.MatMul(input_matrix, w_matrix, &temp_out_tensor);

    // temp_out_tensor.Resize(temp_out_dims);

    Tensor multihead_temp_tensor;
    // B * head_number * S * S * 1 + B * S * 3 * N * H
    int scratch_size = batch * head_number * seq_len * seq_len * 1;
    multihead_temp_tensor.Resize({scratch_size + temp_out_tensor.numel()});
    auto *multihead_temp_data =
        multihead_temp_tensor.mutable_data<T>(context.GetPlace());
    auto *qkptr = multihead_temp_data;
    auto *tptr = multihead_temp_data + scratch_size;

    auto stream = device_ctx.stream();
    // Do the transpose with bias.
    // BxSx3xNxH => tptr: 3xBxNxSxH.
    TransQKVWithBias(batch, seq_len, head_size, head_number, temp_out_data,
                     bias_d, tptr, stream);

    math::MultiHeadGPUComputeFunctor<T> multihead_compute_func;
    multihead_compute_func(device_ctx, batch, seq_len, head_number, head_size,
                           qkptr, bias_qk_d, tptr, scale, T(0.0));

    int grid = batch * head_number * seq_len;
    int block = head_size;
    transpose<T><<<grid, block, 0, stream>>>(tptr, output_d, batch, seq_len,
                                             head_number, head_size);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    multihead_matmul,
    ops::MultiHeadMatMulV2Kernel<paddle::platform::CUDADeviceContext, float>);
