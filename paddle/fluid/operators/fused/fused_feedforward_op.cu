/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/op_version_registry.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/operators/matmul_v2_op.h"

#include "paddle/fluid/operators/fused/fused_dropout_helper.h"
#include "paddle/fluid/operators/layer_norm_kernel.cu.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename DeviceContext, typename T>
class FusedFeedForwardKernel : public framework::OpKernel<T> {
 public:
  void MatMul(const platform::CUDADeviceContext& ctx,
              const framework::Tensor& a, const framework::Tensor& b,
              framework::Tensor* c) const {
    auto blas = math::GetBlas<DeviceContext, T>(ctx);
    auto a_2d = FoldInitDims(a);
    auto b_2d = FoldInitDims(b);
    auto mat_dim_a = math::CreateMatrixDescriptor(a_2d.dims(), 0, false);
    auto mat_dim_b = math::CreateMatrixDescriptor(b_2d.dims(), 0, false);
    T alpha = static_cast<T>(1.0);
    blas.MatMul(a, mat_dim_a, b, mat_dim_b, alpha, c, T(0));
  }

  void FFN(const framework::Tensor& x, const framework::Tensor& linear1_weight,
           const framework::Tensor* linear1_bias,
           const framework::Tensor& linear2_weight,
           const framework::Tensor* linear2_bias,
           const framework::Tensor* ln1_scale,
           const framework::Tensor* ln1_bias,
           const framework::Tensor* ln2_scale,
           const framework::Tensor* ln2_bias, framework::Tensor* out,
           framework::Tensor* dropout1_mask, framework::Tensor* dropout2_mask,
           framework::Tensor* ln1_mean, framework::Tensor* ln1_variance,
           framework::Tensor* ln2_mean, framework::Tensor* ln2_variance,
           framework::Tensor* linear1_out, framework::Tensor* ln1_out,
           framework::Tensor* dropout1_out, framework::Tensor* dropout2_out,
           const int bsz_seq, const int d_model, const int dim_feedforward,
           const std::string& act_method, const bool pre_layer_norm,
           const float epsilon1, const float epsilon2,
           const DropoutParam& dropout_param1,
           const DropoutParam& dropout_param2,
           const platform::CUDADeviceContext& ctx) const {
    FusedDropoutLayerNormHelper<T, uint8_t> pre_layernorm_helper(
        bsz_seq, d_model, epsilon1);
    FusedDropoutHelper<T, uint8_t> fused_act_dropout_helper(
        ctx, bsz_seq, dim_feedforward, dropout_param1);
    FusedDropoutLayerNormHelper<T, uint8_t> fused_dropout_layernorm_helper(
        ctx, bsz_seq, d_model, dropout_param2, epsilon2);

    auto place = ctx.GetPlace();
    using U = LayerNormParamType<T>;
    const framework::Tensor* in = &x;

    const U* ln1_scale_ptr =
        ln1_scale == nullptr ? nullptr : ln1_scale->data<U>();
    const U* ln1_bias_ptr = ln1_bias == nullptr ? nullptr : ln1_bias->data<U>();
    const U* ln2_scale_ptr =
        ln2_scale == nullptr ? nullptr : ln2_scale->data<U>();
    const U* ln2_bias_ptr = ln2_bias == nullptr ? nullptr : ln2_bias->data<U>();
    const T* linear1_bias_ptr =
        linear1_bias == nullptr ? nullptr : linear1_bias->data<T>();
    const T* linear2_bias_ptr =
        linear2_bias == nullptr ? nullptr : linear2_bias->data<T>();

    if (pre_layer_norm) {
      pre_layernorm_helper.LayerNorm(
          ctx, x.data<T>(), ln1_scale_ptr, ln1_bias_ptr, ln1_out->data<T>(),
          ln1_mean->data<U>(), ln1_variance->data<U>());
      in = ln1_out;
    }
    MatMul(ctx, *in, linear1_weight, linear1_out);
    fused_act_dropout_helper.DropoutActBias(
        ctx, linear1_out->data<T>(), linear1_bias_ptr, act_method,
        dropout1_out->data<T>(), dropout1_mask->data<uint8_t>());
    framework::Tensor linear2_out;
    linear2_out.mutable_data<T>({bsz_seq, d_model}, place);
    MatMul(ctx, *dropout1_out, linear2_weight, &linear2_out);
    if (!pre_layer_norm) {
      fused_dropout_layernorm_helper.LayernormResidualDropoutBias(
          ctx, linear2_out.data<T>(), x.data<T>(), linear2_bias_ptr,
          ln2_scale_ptr, ln2_bias_ptr, dropout2_out->data<T>(),
          dropout2_mask->data<uint8_t>(), out->data<T>(), ln2_mean->data<U>(),
          ln2_variance->data<U>());
    } else {
      fused_dropout_layernorm_helper.ResidualDropoutBias(
          ctx, linear2_out.data<T>(), x.data<T>(), linear2_bias_ptr,
          out->data<T>(), dropout2_mask->data<uint8_t>());
    }
  }

  void Compute(const framework::ExecutionContext& context) const override {
    auto* x = context.Input<framework::Tensor>("X");
    auto* linear1_weight = context.Input<framework::Tensor>("Linear1Weight");
    auto* linear1_bias = context.Input<framework::Tensor>("Linear1Bias");
    auto* linear2_weight = context.Input<framework::Tensor>("Linear2Weight");
    auto* linear2_bias = context.Input<framework::Tensor>("Linear2Bias");
    auto* ln1_scale = context.Input<framework::Tensor>("Ln1Scale");
    auto* ln1_bias = context.Input<framework::Tensor>("Ln1Bias");
    auto* ln2_scale = context.Input<framework::Tensor>("Ln2Scale");
    auto* ln2_bias = context.Input<framework::Tensor>("Ln2Bias");

    auto* ln1_mean = context.Output<framework::Tensor>("Ln1Mean");
    auto* ln1_variance = context.Output<framework::Tensor>("Ln1Variance");
    auto* ln2_mean = context.Output<framework::Tensor>("Ln2Mean");
    auto* ln2_variance = context.Output<framework::Tensor>("Ln2Variance");
    auto* out = context.Output<framework::Tensor>("Out");
    auto* dropout1_mask = context.Output<framework::Tensor>("Dropout1Mask");
    auto* dropout2_mask = context.Output<framework::Tensor>("Dropout2Mask");
    auto* linear1_out = context.Output<framework::Tensor>("Linear1Out");
    auto* ln1_out = context.Output<framework::Tensor>("Ln1Out");
    auto* dropout1_out = context.Output<framework::Tensor>("Dropout1Out");
    auto* dropout2_out = context.Output<framework::Tensor>("Dropout2Out");

    const std::string act_method = context.Attr<std::string>("act_method");

    const bool pre_layer_norm = context.Attr<bool>("pre_layer_norm");
    const float epsilon1 = context.Attr<float>("ln1_epsilon");
    const float epsilon2 = context.Attr<float>("ln2_epsilon");

    DropoutParam dropout_param1(context, 1);
    DropoutParam dropout_param2(context, 2);

    using U = LayerNormParamType<T>;
    auto place = context.GetPlace();
    out->mutable_data<T>(place);
    dropout1_mask->mutable_data<uint8_t>(place);
    dropout2_mask->mutable_data<uint8_t>(place);
    ln1_mean->mutable_data<U>(place);
    ln1_variance->mutable_data<U>(place);
    ln2_mean->mutable_data<U>(place);
    ln2_variance->mutable_data<U>(place);
    linear1_out->mutable_data<T>(place);
    ln1_out->mutable_data<T>(place);
    dropout1_out->mutable_data<T>(place);
    dropout2_out->mutable_data<T>(place);

    auto x_dim = x->dims();
    auto mat_dim_x =
        math::CreateMatrixDescriptor(RowMatrixFromVector(x_dim), 0, false);

    auto dim = linear1_weight->dims();
    int d_model = dim[0];
    int dim_feedforward = dim[dim.size() - 1];
    int bsz_seq = mat_dim_x.batch_size_ * mat_dim_x.height_;

    FFN(*x, *linear1_weight, linear1_bias, *linear2_weight, linear2_bias,
        ln1_scale, ln1_bias, ln2_scale, ln2_bias, out, dropout1_mask,
        dropout2_mask, ln1_mean, ln1_variance, ln2_mean, ln2_variance,
        linear1_out, ln1_out, dropout1_out, dropout2_out, bsz_seq, d_model,
        dim_feedforward, act_method, pre_layer_norm, epsilon1, epsilon2,
        dropout_param1, dropout_param2, context.cuda_device_context());
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fused_feedforward,
    ops::FusedFeedForwardKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FusedFeedForwardKernel<paddle::platform::CUDADeviceContext, double>,
    ops::FusedFeedForwardKernel<paddle::platform::CUDADeviceContext,
                                paddle::platform::float16>);
