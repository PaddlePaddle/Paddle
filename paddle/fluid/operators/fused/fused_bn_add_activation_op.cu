#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/operators/fused/fused_bn_add_activation_op.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/norm_utils.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"

DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace paddle {
namespace operators {
using Tensor = framework::Tensor;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
class FusedBatchNormAddActKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    float momentum = ctx.Attr<float>("momentum");
    std::string act_type = ctx.Attr<std::string>("act_type");

    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    // Get the size for each dimension.
    // NHWC [batch_size, in_height, in_width, in_channels]
    const auto *x = ctx.Input<Tensor>("X");
    const auto *z = ctx.Input<Tensor>("Z");
    const auto &in_dims = x->dims();

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    auto *mean_out = ctx.Output<Tensor>("MeanOut");
    auto *variance_out = ctx.Output<Tensor>("VarianceOut");
    mean_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    variance_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

    auto *saved_mean = ctx.Output<Tensor>("SavedMean");
    auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
    saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

    auto *y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    int N, C, H, W, D;
    const DataLayout data_layout = DataLayout::kNHWC;
    ExtractNCWHD(in_dims, data_layout, &N, &C, &H, &W, &D);

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    // ------------------- cudnn descriptors ---------------------
    auto handle = dev_ctx.cudnn_handle();
    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t bn_param_desc_;
    hipdnnBatchNormMode_t mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));

    std::vector<int> dims = {N, C, H, W, D};
    std::vector<int> strides = {H * W * D * C, 1, W * D * C, D * C, C};

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        in_dims.size() > 3 ? in_dims.size() : 4, dims.data(), strides.data()));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDeriveBNTensorDescriptor(bn_param_desc_,
                                                         data_desc_, mode_));

    double this_factor = 1. - momentum;
    cudnnBatchNormOps_t bnOps_ = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    platform::ScopedActivationDescriptor scope_act_desc;
    hipdnnActivationDescriptor_t activation_desc_ =
        scope_act_desc.descriptor<T>(act_type);
    size_t workspace_size = 0;
    size_t reserve_space_size = 0;
    void *reserve_space_ptr = nullptr;
    void *workspace_ptr = nullptr;
    Tensor workspace_tensor;
    // Create reserve space and workspace for batch norm.
    // Create tensor for each batchnorm op, it will be used in the
    // backward. Thus this tensor shouldn't be temp.
    auto *reserve_space = ctx.Output<Tensor>("ReserveSpace");
    PADDLE_ENFORCE_NOT_NULL(
        reserve_space,
        platform::errors::NotFound(
            "The argument ReserveSpace of batch_norm op is not found."));

    // --------------- cudnn batchnorm workspace ---------------
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::
            cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
                /*handle=*/handle,
                /*mode=*/mode_,
                /*bnOps=*/bnOps_,
                /*xDesc=*/data_desc_,
                /*zDesc=*/data_desc_,
                /*yDesc=*/data_desc_,
                /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                /*activationDesc=*/activation_desc_,
                /*sizeInBytes=*/&workspace_size));

    // -------------- cudnn batchnorm reserve space --------------
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
            /*handle=*/handle,
            /*mode=*/mode_,
            /*bnOps=*/bnOps_,
            /*activationDesc=*/activation_desc_,
            /*xDesc=*/data_desc_,
            /*sizeInBytes=*/&reserve_space_size));

    reserve_space_ptr = reserve_space->mutable_data(ctx.GetPlace(), x->type(),
                                                    reserve_space_size);
    workspace_ptr = workspace_tensor.mutable_data(ctx.GetPlace(), x->type(),
                                                  workspace_size);
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::cudnnBatchNormalizationForwardTrainingEx(
            handle, mode_, bnOps_, CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(), data_desc_, x->template data<T>(),
            data_desc_, z->template data<T>(), data_desc_,
            y->template data<T>(), bn_param_desc_,
            scale->template data<BatchNormParamType<T>>(),
            bias->template data<BatchNormParamType<T>>(), this_factor,
            mean_out->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace()),
            variance_out->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace()),
            epsilon, saved_mean->template mutable_data<BatchNormParamType<T>>(
                         ctx.GetPlace()),
            saved_variance->template mutable_data<BatchNormParamType<T>>(
                ctx.GetPlace()),
            activation_desc_, workspace_ptr, workspace_size, reserve_space_ptr,
            reserve_space_size));

    // clean when exit.
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
  }
};

template <typename T>
class FusedBatchNormAddActGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    std::string act_type = ctx.Attr<std::string>("act_type");

    const auto *x = ctx.Input<Tensor>("X");
    const auto *y = ctx.Input<Tensor>("Y");
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");
    const auto *reserve_space = ctx.Input<Tensor>("ReserveSpace");

    const auto &in_dims = x->dims();

    int N, C, H, W, D;
    const DataLayout data_layout = DataLayout::kNHWC;
    ExtractNCWHD(in_dims, data_layout, &N, &C, &H, &W, &D);

    // init output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_z = ctx.Output<Tensor>(framework::GradVarName("Z"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    d_x->mutable_data<T>(ctx.GetPlace());
    d_z->mutable_data<T>(ctx.GetPlace());
    PADDLE_ENFORCE_EQ(
        d_scale && d_bias, true,
        platform::errors::PreconditionNotMet(
            "Both the scale grad and the bias grad must not be null."));
    d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    PADDLE_ENFORCE_EQ(scale->dims().size(), 1UL,
                      platform::errors::PreconditionNotMet(
                          "The scale only has one dimension."));
    PADDLE_ENFORCE_EQ(
        scale->dims()[0], C,
        platform::errors::PreconditionNotMet(
            "The size of scale is equal to the channel of Input(X)."));

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    std::vector<int> dims = {N, C, H, W, D};
    std::vector<int> strides = {H * W * C * D, 1, W * D * C, D * C, C};
    // ------------------- cudnn descriptors ---------------------
    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t bn_param_desc_;
    hipdnnBatchNormMode_t mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;

    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        in_dims.size() > 3 ? in_dims.size() : 4, dims.data(), strides.data()));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDeriveBNTensorDescriptor(bn_param_desc_,
                                                         data_desc_, mode_));

    const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
    const auto *saved_mean_data =
        saved_mean->template data<BatchNormParamType<T>>();
    const auto *saved_var_data =
        saved_var->template data<BatchNormParamType<T>>();

    size_t workspace_size = 0;
    void *workspace_ptr = nullptr;
    Tensor workspace_tensor;
    auto reserve_space_size = reserve_space->memory_size();
    cudnnBatchNormOps_t bnOps_ = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    platform::ScopedActivationDescriptor scope_act_desc;
    hipdnnActivationDescriptor_t activation_desc_ =
        scope_act_desc.descriptor<T>(act_type);
    // --------------- cudnn batchnorm workspace ---------------
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::cudnnGetBatchNormalizationBackwardExWorkspaceSize(
            /*handle=*/dev_ctx.cudnn_handle(),
            /*mode=*/mode_,
            /*bnOps=*/bnOps_,
            /*xDesc=*/data_desc_,
            /*yDesc=*/data_desc_,
            /*dyDesc=*/data_desc_,
            /*dzDesc=*/data_desc_,
            /*dxDesc=*/data_desc_,
            /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
            /*activationDesc=*/activation_desc_,
            /*sizeInBytes=*/&workspace_size));

    workspace_ptr = workspace_tensor.mutable_data(ctx.GetPlace(), x->type(),
                                                  workspace_size);
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::cudnnBatchNormalizationBackwardEx(
            /*handle=*/dev_ctx.cudnn_handle(),
            /*mode=*/mode_,
            /*bnOps=*/bnOps_,
            /*alphaDataDiff=*/CudnnDataType<T>::kOne(),
            /*betaDataDiff=*/CudnnDataType<T>::kZero(),
            /*alphaParamDiff=*/CudnnDataType<T>::kOne(),
            /*betaParamDiff=*/CudnnDataType<T>::kZero(),
            /*xDesc=*/data_desc_,
            /*xData=*/x->template data<T>(),
            /*yDesc=*/data_desc_,
            /*yData=*/y->template data<T>(),
            /*dyDesc=*/data_desc_,
            /*dyData=*/d_y->template data<T>(),
            /*dzDesc=*/data_desc_,
            /*dzData=*/d_z->template data<T>(),
            /*dxDesc=*/data_desc_,
            /*dxData=*/d_x->template data<T>(),
            /*dBnScaleBiasDesc=*/bn_param_desc_,
            /*bnScaleData=*/scale->template data<BatchNormParamType<T>>(),
            /*bnBiasData=*/bias->template data<BatchNormParamType<T>>(),
            /*dBnScaleData=*/d_scale->template data<BatchNormParamType<T>>(),
            /*dBnBiasData=*/d_bias->template data<BatchNormParamType<T>>(),
            /*epsilon=*/epsilon,
            /*savedMean=*/saved_mean_data,
            /*savedInvVariance=*/saved_var_data,
            /*activationDesmc=*/activation_desc_,
            /*workspace=*/workspace_ptr,
            /*workSpaceSizeInBytes=*/workspace_size,
            /*reserveSpace=*/const_cast<T *>(reserve_space->template data<T>()),
            /*reserveSpaceSizeInBytes=*/reserve_space_size));

    // clean when exit.
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
  }
};

}  // namespace operators
}  // namespace paddle

#if CUDNN_VERSION >= 7401
namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    fused_bn_add_activation,
    ops::FusedBatchNormAddActKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(fused_bn_add_activation_grad,
                        ops::FusedBatchNormAddActGradKernel<
                            plat::CUDADeviceContext, plat::float16>);
#endif
