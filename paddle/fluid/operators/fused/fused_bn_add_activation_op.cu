#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/framework/scope_guard.h"
#include "paddle/fluid/operators/activation_op.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_impl.cu.h"
#include "paddle/fluid/operators/fused/fused_bn_add_activation_op.h"
#include "paddle/fluid/operators/norm_utils.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/pten/kernels/funcs/math_function.h"

DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace paddle {
namespace operators {

template <typename T>
struct AddReluFunctor {
  HOSTDEVICE T operator()(T x, T y) const {
    T z = x + y;
    return z > static_cast<T>(0) ? z : static_cast<T>(0);
  }
};

using Tensor = framework::Tensor;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T>
class FusedBatchNormAddActKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    float momentum = ctx.Attr<float>("momentum");
    std::string act_type = ctx.Attr<std::string>("act_type");
    const DataLayout data_layout =
        framework::StringToDataLayout(ctx.Attr<std::string>("data_layout"));

    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    // Get the size for each dimension.
    // NHWC [batch_size, in_height, in_width, in_channels]
    const auto *x = ctx.Input<Tensor>("X");
    const auto *x_data = x->template data<T>();
    const auto *z = ctx.Input<Tensor>("Z");
    const auto *z_data = z->template data<T>();
    const auto &in_dims = x->dims();

    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *scale_data = scale->template data<BatchNormParamType<T>>();
    const auto *bias = ctx.Input<Tensor>("Bias");
    const auto *bias_data = bias->template data<BatchNormParamType<T>>();

    auto *mean_out = ctx.Output<Tensor>("MeanOut");
    auto *variance_out = ctx.Output<Tensor>("VarianceOut");
    auto *mean_out_data =
        mean_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    auto *variance_out_data =
        variance_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

    auto *saved_mean = ctx.Output<Tensor>("SavedMean");
    auto *saved_variance = ctx.Output<Tensor>("SavedVariance");
    auto *saved_mean_data =
        saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    auto *saved_variance_data =
        saved_variance->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

    auto *y = ctx.Output<Tensor>("Y");
    auto *y_data = y->mutable_data<T>(ctx.GetPlace());

    int N, C, H, W, D;
    ExtractNCWHD(in_dims, data_layout, &N, &C, &H, &W, &D);

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    // ------------------- cudnn descriptors ---------------------
    auto handle = dev_ctx.cudnn_handle();
    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t bn_param_desc_;
    hipdnnBatchNormMode_t mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;

    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));

    std::vector<int> dims, strides;
    if (data_layout == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * D * C, 1, W * D * C, D * C, C};
    }

    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        in_dims.size() > 3 ? in_dims.size() : 4, dims.data(), strides.data()));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnDeriveBNTensorDescriptor(
        bn_param_desc_, data_desc_, mode_));

    DEFINE_PADDLE_SCOPE_GUARD([=] {
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
    });

    double this_factor = 1. - momentum;
    platform::ScopedActivationDescriptor scope_act_desc;
    cudnnBatchNormOps_t bnOps_;
    hipdnnActivationDescriptor_t activation_desc_;
    if (std::is_same<T, platform::float16>::value) {
      bnOps_ = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
      activation_desc_ = scope_act_desc.descriptor<T>(act_type);
    } else {
      bnOps_ = CUDNN_BATCHNORM_OPS_BN;
      activation_desc_ = nullptr;
    }

    size_t workspace_size = 0;
    size_t reserve_space_size = 0;
    void *reserve_space_ptr = nullptr;
    void *workspace_ptr = nullptr;
    Tensor workspace_tensor;
    // Create reserve space and workspace for batch norm.
    // Create tensor for each batchnorm op, it will be used in the
    // backward. Thus this tensor shouldn't be temp.
    auto *reserve_space = ctx.Output<Tensor>("ReserveSpace");
    PADDLE_ENFORCE_NOT_NULL(
        reserve_space,
        platform::errors::NotFound(
            "The argument ReserveSpace of batch_norm op is not found."));

    // --------------- cudnn batchnorm workspace ---------------
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::
            cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
                /*handle=*/handle,
                /*mode=*/mode_,
                /*bnOps=*/bnOps_,
                /*xDesc=*/data_desc_,
                /*zDesc=*/data_desc_,
                /*yDesc=*/data_desc_,
                /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                /*activationDesc=*/activation_desc_,
                /*sizeInBytes=*/&workspace_size));

    // -------------- cudnn batchnorm reserve space --------------
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
            /*handle=*/handle,
            /*mode=*/mode_,
            /*bnOps=*/bnOps_,
            /*activationDesc=*/activation_desc_,
            /*xDesc=*/data_desc_,
            /*sizeInBytes=*/&reserve_space_size));

    reserve_space_ptr = reserve_space->mutable_data(ctx.GetPlace(), x->dtype(),
                                                    reserve_space_size);
    workspace_ptr = workspace_tensor.mutable_data(ctx.GetPlace(), x->dtype(),
                                                  workspace_size);

    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::cudnnBatchNormalizationForwardTrainingEx(
            handle, mode_, bnOps_, CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(), data_desc_, x_data, data_desc_, z_data,
            data_desc_, y_data, bn_param_desc_, scale_data, bias_data,
            this_factor, mean_out_data, variance_out_data, epsilon,
            saved_mean_data, saved_variance_data, activation_desc_,
            workspace_ptr, workspace_size, reserve_space_ptr,
            reserve_space_size));
    if (!std::is_same<T, platform::float16>::value) {
      // add + relu
      std::vector<const framework::Tensor *> ins = {y, z};
      std::vector<framework::Tensor *> outs = {y};
      PADDLE_ENFORCE_EQ(act_type, "relu", platform::errors::InvalidArgument(
                                              "The act_type must be relu."));
      pten::funcs::LaunchSameDimsElementwiseCudaKernel<T, AddReluFunctor<T>, 1>(
          dev_ctx, ins, &outs, AddReluFunctor<T>());
    }
  }
};

template <typename T>
class FusedBatchNormAddActGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::PreconditionNotMet("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    std::string act_type = ctx.Attr<std::string>("act_type");
    const DataLayout data_layout =
        framework::StringToDataLayout(ctx.Attr<std::string>("data_layout"));

    const auto *x = ctx.Input<Tensor>("X");
    const auto *x_data = x->template data<T>();
    const auto *y = ctx.Input<Tensor>("Y");
    const auto *y_data = y->template data<T>();
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *dy_data = d_y->template data<T>();
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *scale_data = scale->template data<BatchNormParamType<T>>();

    const auto *bias = ctx.Input<Tensor>("Bias");
    const auto *bias_data = bias->template data<BatchNormParamType<T>>();
    const auto *reserve_space = ctx.Input<Tensor>("ReserveSpace");

    const auto &in_dims = x->dims();

    int N, C, H, W, D;
    ExtractNCWHD(in_dims, data_layout, &N, &C, &H, &W, &D);

    // init output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_z = ctx.Output<Tensor>(framework::GradVarName("Z"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    auto *dx_data = d_x->mutable_data<T>(ctx.GetPlace());
    auto *dz_data = d_z->mutable_data<T>(ctx.GetPlace());
    PADDLE_ENFORCE_EQ(
        d_scale && d_bias, true,
        platform::errors::PreconditionNotMet(
            "Both the scale grad and the bias grad must not be null."));
    auto *dscale_data =
        d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    auto *dbias_data =
        d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    PADDLE_ENFORCE_EQ(scale->dims().size(), 1UL,
                      platform::errors::PreconditionNotMet(
                          "The scale only has one dimension."));
    PADDLE_ENFORCE_EQ(
        scale->dims()[0], C,
        platform::errors::PreconditionNotMet(
            "The size of scale is equal to the channel of Input(X)."));

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    std::vector<int> dims, strides;
    if (data_layout == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * D * C, 1, W * D * C, D * C, C};
    }
    // ------------------- cudnn descriptors ---------------------
    hipdnnTensorDescriptor_t data_desc_;
    hipdnnTensorDescriptor_t bn_param_desc_;
    hipdnnBatchNormMode_t mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;

    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
    if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than "
                 << "HIPDNN_BN_MIN_EPSILON. Setting it to "
                 << "HIPDNN_BN_MIN_EPSILON instead.";
    }
    epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        data_desc_, CudnnDataType<T>::type,
        in_dims.size() > 3 ? in_dims.size() : 4, dims.data(), strides.data()));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnDeriveBNTensorDescriptor(
        bn_param_desc_, data_desc_, mode_));

    const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *saved_var = ctx.Input<Tensor>("SavedVariance");
    const auto *saved_mean_data =
        saved_mean->template data<BatchNormParamType<T>>();
    const auto *saved_var_data =
        saved_var->template data<BatchNormParamType<T>>();

    DEFINE_PADDLE_SCOPE_GUARD([=] {
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
    });

    auto handle = dev_ctx.cudnn_handle();
    size_t workspace_size = 0;
    void *workspace_ptr = nullptr;
    Tensor workspace_tensor;
    auto reserve_space_size = reserve_space->memory_size();
    cudnnBatchNormOps_t bnOps_;
    platform::ScopedActivationDescriptor scope_act_desc;
    hipdnnActivationDescriptor_t activation_desc_ =
        scope_act_desc.descriptor<T>(act_type);
    if (std::is_same<T, platform::float16>::value) {
      bnOps_ = CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION;
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipdnnActivationBackward(
          handle, activation_desc_, CudnnDataType<T>::kOne(), data_desc_,
          y_data, data_desc_, dy_data, data_desc_, y_data,
          CudnnDataType<T>::kZero(), data_desc_, dz_data));

      bnOps_ = CUDNN_BATCHNORM_OPS_BN;
      activation_desc_ = nullptr;
      dy_data = dz_data;
      dz_data = nullptr;
    }
    // --------------- cudnn batchnorm workspace ---------------
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::cudnnGetBatchNormalizationBackwardExWorkspaceSize(
            /*handle=*/handle,
            /*mode=*/mode_,
            /*bnOps=*/bnOps_,
            /*xDesc=*/data_desc_,
            /*yDesc=*/data_desc_,
            /*dyDesc=*/data_desc_,
            /*dzDesc=*/data_desc_,
            /*dxDesc=*/data_desc_,
            /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
            /*activationDesc=*/activation_desc_,
            /*sizeInBytes=*/&workspace_size));

    workspace_ptr = workspace_tensor.mutable_data(ctx.GetPlace(), x->dtype(),
                                                  workspace_size);
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::cudnnBatchNormalizationBackwardEx(
            /*handle=*/handle,
            /*mode=*/mode_,
            /*bnOps=*/bnOps_,
            /*alphaDataDiff=*/CudnnDataType<T>::kOne(),
            /*betaDataDiff=*/CudnnDataType<T>::kZero(),
            /*alphaParamDiff=*/CudnnDataType<T>::kOne(),
            /*betaParamDiff=*/CudnnDataType<T>::kZero(),
            /*xDesc=*/data_desc_,
            /*xData=*/x_data,
            /*yDesc=*/data_desc_,
            /*yData=*/y_data,
            /*dyDesc=*/data_desc_,
            /*dyData=*/dy_data,
            /*dzDesc=*/data_desc_,
            /*dzData=*/dz_data,
            /*dxDesc=*/data_desc_,
            /*dxData=*/dx_data,
            /*dBnScaleBiasDesc=*/bn_param_desc_,
            /*bnScaleData=*/scale_data,
            /*bnBiasData=*/bias_data,
            /*dBnScaleData=*/dscale_data,
            /*dBnBiasData=*/dbias_data,
            /*epsilon=*/epsilon,
            /*savedMean=*/saved_mean_data,
            /*savedInvVariance=*/saved_var_data,
            /*activationDesmc=*/activation_desc_,
            /*workspace=*/workspace_ptr,
            /*workSpaceSizeInBytes=*/workspace_size,
            /*reserveSpace=*/const_cast<T *>(reserve_space->template data<T>()),
            /*reserveSpaceSizeInBytes=*/reserve_space_size));
  }
};

}  // namespace operators
}  // namespace paddle

#if CUDNN_VERSION >= 7401
namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    fused_bn_add_activation,
    ops::FusedBatchNormAddActKernel<plat::CUDADeviceContext, plat::float16>,
    ops::FusedBatchNormAddActKernel<plat::CUDADeviceContext, float>);
REGISTER_OP_CUDA_KERNEL(
    fused_bn_add_activation_grad,
    ops::FusedBatchNormAddActGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::FusedBatchNormAddActGradKernel<plat::CUDADeviceContext, float>);
#endif
