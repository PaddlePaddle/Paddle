#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/tensor.h"

namespace paddle {
namespace operators {

typedef struct { float x, y, w, h; } box;

typedef struct detection {
  box bbox;
  int classes;
  float* prob;
  float* mask;
  float objectness;
  int sort_class;
  int max_prob_class_index;
} detection;

typedef struct TensorInfo {
  int bbox_count_host;  // record bbox numbers
  int bbox_count_max_alloc{50};
  float* bboxes_dev_ptr;
  float* bboxes_host_ptr;
  int* bbox_count_device_ptr;  // box counter in gpu memory, used by atomicAdd
} TensorInfo;

__global__ void YoloBoxNum(const float* input, int* bbox_count,
                           const uint grid_size, const uint class_num,
                           const uint anchors_num, float prob_thresh) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;
  if ((x_id >= grid_size) || (y_id >= grid_size) || (z_id >= anchors_num)) {
    return;
  }

  const int grids_num = grid_size * grid_size;
  const int bbindex = y_id * grid_size + x_id;
  // objectness
  float objectness = input[bbindex + grids_num * (z_id * (5 + class_num) + 4)];
  if (objectness < prob_thresh) {
    return;
  }

  atomicAdd(bbox_count, 1);
}

__global__ void YoloTensorParseKernel(
    const float* input, const float* im_shape_data, const float* im_scale_data,
    float* output, int* bbox_index, const uint grid_size, const uint class_num,
    const uint anchors_num, const uint netw, const uint neth, int* biases,
    float prob_thresh) {
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if ((x_id >= grid_size) || (y_id >= grid_size) || (z_id >= anchors_num)) {
    return;
  }

  const float pic_h = im_shape_data[0] / im_scale_data[0];
  const float pic_w = im_shape_data[1] / im_scale_data[1];

  const int grids_num = grid_size * grid_size;
  const int bbindex = y_id * grid_size + x_id;

  // objectness
  float objectness = input[bbindex + grids_num * (z_id * (5 + class_num) + 4)];
  if (objectness < prob_thresh) {
    return;
  }

  int cur_bbox_index = atomicAdd(bbox_index, 1);
  int tensor_index = cur_bbox_index * (5 + class_num);

  // x
  float x = input[bbindex + grids_num * (z_id * (5 + class_num) + 0)];
  x = (x + static_cast<float>(x_id)) * static_cast<float>(pic_w) /
      static_cast<float>(grid_size);
  // y
  float y = input[bbindex + grids_num * (z_id * (5 + class_num) + 1)];
  y = (y + static_cast<float>(y_id)) * static_cast<float>(pic_h) /
      static_cast<float>(grid_size);
  // w
  float w = input[bbindex + grids_num * (z_id * (5 + class_num) + 2)];
  w = w * biases[2 * z_id] * pic_w / netw;
  // h
  float h = input[bbindex + grids_num * (z_id * (5 + class_num) + 3)];
  h = h * biases[2 * z_id + 1] * pic_h / neth;

  // CorrectYoloBox(x, y, w, h, pic_w, pic_h, netw, neth);
  output[tensor_index] = objectness;
  output[tensor_index + 1] = x - w / 2;
  output[tensor_index + 2] = y - h / 2;
  output[tensor_index + 3] = x + w / 2;
  output[tensor_index + 4] = y + h / 2;
  output[tensor_index + 1] =
      output[tensor_index + 1] > 0 ? output[tensor_index + 1] : 0.f;
  output[tensor_index + 2] =
      output[tensor_index + 2] > 0 ? output[tensor_index + 2] : 0.f;
  output[tensor_index + 3] = output[tensor_index + 3] < pic_w - 1
                                 ? output[tensor_index + 3]
                                 : pic_w - 1;
  output[tensor_index + 4] = output[tensor_index + 4] < pic_h - 1
                                 ? output[tensor_index + 4]
                                 : pic_h - 1;

  // Probabilities of classes
  for (uint i = 0; i < class_num; ++i) {
    float prob =
        input[bbindex + grids_num * (z_id * (5 + class_num) + (5 + i))] *
        objectness;
    output[tensor_index + 5 + i] = prob;
  }
}

static int nms_comparator(const void* pa, const void* pb) {
  const detection a = *reinterpret_cast<const detection*>(pa);
  const detection b = *reinterpret_cast<const detection*>(pb);
  float diff = 0;

  if (a.max_prob_class_index > b.max_prob_class_index)
    return 1;
  else if (a.max_prob_class_index < b.max_prob_class_index)
    return -1;

  if (b.sort_class >= 0) {
    diff = a.prob[b.sort_class] - b.prob[b.sort_class];
  } else {
    diff = a.objectness - b.objectness;
  }

  if (diff < 0)
    return 1;
  else if (diff > 0)
    return -1;
  return 0;
}

static float overlap(float x1, float w1, float x2, float w2) {
  float l1 = x1 - w1 / 2;
  float l2 = x2 - w2 / 2;
  float left = l1 > l2 ? l1 : l2;
  float r1 = x1 + w1 / 2;
  float r2 = x2 + w2 / 2;
  float right = r1 < r2 ? r1 : r2;
  return right - left;
}

static float box_intersection(box a, box b) {
  float w = overlap(a.x, a.w, b.x, b.w);
  float h = overlap(a.y, a.h, b.y, b.h);
  if (w < 0 || h < 0) return 0;
  float area = w * h;
  return area;
}

static float box_union(box a, box b) {
  float i = box_intersection(a, b);
  float u = a.w * a.h + b.w * b.h - i;
  return u;
}

static float box_iou(box a, box b) {
  return box_intersection(a, b) / box_union(a, b);
}

static void post_nms(std::vector<detection>* det_bboxes, float thresh,
                     int classes) {
  int total = det_bboxes->size();
  if (total <= 0) {
    return;
  }

  detection* dets = det_bboxes->data();

  int i, j, k;
  k = total - 1;
  for (i = 0; i <= k; ++i) {
    if (dets[i].objectness == 0) {
      detection swap = dets[i];
      dets[i] = dets[k];
      dets[k] = swap;
      --k;
      --i;
    }
  }
  total = k + 1;

  qsort(dets, total, sizeof(detection), nms_comparator);

  for (i = 0; i < total; ++i) {
    if (dets[i].objectness == 0) {
      continue;
    }

    box a = dets[i].bbox;

    for (j = i + 1; j < total; ++j) {
      if (dets[j].objectness == 0) {
        continue;
      }
      if (dets[j].max_prob_class_index != dets[i].max_prob_class_index) break;

      box b = dets[j].bbox;

      if (box_iou(a, b) > thresh) {
        dets[j].objectness = 0;
        for (k = 0; k < classes; ++k) {
          dets[j].prob[k] = 0;
        }
      }
    }
  }
}

static void YoloTensorParseCuda(
    const float* input_data,  // [in] YOLO_BOX_HEAD layer output
    const float* image_shape_data, const float* image_scale_data,
    float** bboxes_tensor_ptr,  // [out] Bounding boxes output tensor
    int* bbox_count_max_alloc,  // [in/out] maximum bounding box number
                                // allocated in dev
    int* bbox_count_host,  // [in/out] bounding boxes number recorded in host
    int* bbox_count_device_ptr,  // [in/out] bounding boxes number calculated
                                 // in
                                 // device side
    int* bbox_index_device_ptr,  // [in] bounding box index for kernel threads
                                 // shared access
    int grid_size, int class_num, int anchors_num, int netw, int neth,
    int* biases_device, float prob_thresh) {
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((grid_size / threads_per_block.x) + 1,
                        (grid_size / threads_per_block.y) + 1,
                        (anchors_num / threads_per_block.z) + 1);

  // Estimate how many boxes will be choosed
  int bbox_count = 0;
  hipMemcpy(bbox_count_device_ptr, &bbox_count, sizeof(int),
             hipMemcpyHostToDevice);
  YoloBoxNum<<<number_of_blocks, threads_per_block, 0>>>(
      input_data, bbox_count_device_ptr, grid_size, class_num, anchors_num,
      prob_thresh);
  hipMemcpy(&bbox_count, bbox_count_device_ptr, sizeof(int),
             hipMemcpyDeviceToHost);

  // Record actual bbox number
  *bbox_count_host = bbox_count;

  // Obtain previous allocated bbox tensor in device side
  float* bbox_tensor = *bboxes_tensor_ptr;
  // Update previous maximum bbox number
  if (bbox_count > *bbox_count_max_alloc) {
    hipFree(bbox_tensor);
    hipMalloc(&bbox_tensor, bbox_count * (5 + class_num) * sizeof(float));
    *bbox_count_max_alloc = bbox_count;
    *bboxes_tensor_ptr = bbox_tensor;
  }

  // Now generate bboxes
  int bbox_index = 0;
  hipMemcpy(bbox_index_device_ptr, &bbox_index, sizeof(int),
             hipMemcpyHostToDevice);
  YoloTensorParseKernel<<<number_of_blocks, threads_per_block, 0>>>(
      input_data, image_shape_data, image_scale_data, bbox_tensor,
      bbox_index_device_ptr, grid_size, class_num, anchors_num, netw, neth,
      biases_device, prob_thresh);
}

class YoloBoxPostKernel : public framework::OpKernel<float> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    using Tensor = framework::Tensor;
    // prepare inputs
    std::vector<const float*> boxes_input(3);
    std::vector<std::vector<int32_t>> boxes_input_dims(3);
    for (int i = 0; i < 3; i++) {
      auto* boxes_tensor =
          context.Input<framework::Tensor>("Boxes" + std::to_string(i));
      boxes_input[i] = boxes_tensor->data<float>();
      auto dims = boxes_tensor->dims();
      for (int j = 0; j < dims.size(); j++) {
        boxes_input_dims[i].push_back(dims[j]);
      }
    }
    const float* image_shape_data =
        context.Input<framework::Tensor>("ImageShape")->data<float>();
    const float* image_scale_data =
        context.Input<framework::Tensor>("ImageScale")->data<float>();

    // prepare outputs
    auto* boxes_scores_tensor = context.Output<framework::Tensor>("Out");
    auto* boxes_num_tensor = context.Output<framework::Tensor>("NmsRoisNum");

    // prepare anchors
    std::vector<int32_t> anchors;
    auto anchors0 = context.Attr<std::vector<int>>("anchors0");
    auto anchors1 = context.Attr<std::vector<int>>("anchors1");
    auto anchors2 = context.Attr<std::vector<int>>("anchors2");
    anchors.insert(anchors.end(), anchors0.begin(), anchors0.end());
    anchors.insert(anchors.end(), anchors1.begin(), anchors1.end());
    anchors.insert(anchors.end(), anchors2.begin(), anchors2.end());
    int* device_anchors;
    hipMalloc(reinterpret_cast<void**>(&device_anchors),
               anchors.size() * sizeof(int));
    hipMemcpy(device_anchors, anchors.data(), anchors.size() * sizeof(int),
               hipMemcpyHostToDevice);
    int* device_anchors_ptr[3];
    device_anchors_ptr[0] = device_anchors;
    device_anchors_ptr[1] = device_anchors_ptr[0] + anchors0.size();
    device_anchors_ptr[2] = device_anchors_ptr[1] + anchors1.size();
    std::vector<int> anchors_num{static_cast<int>(anchors0.size()) / 2,
                                 static_cast<int>(anchors1.size()) / 2,
                                 static_cast<int>(anchors2.size()) / 2};

    // prepare other attrs
    int class_num = context.Attr<int>("class_num");
    float conf_thresh = context.Attr<float>("conf_thresh");
    std::vector<int> downsample_ratio{context.Attr<int>("downsample_ratio0"),
                                      context.Attr<int>("downsample_ratio1"),
                                      context.Attr<int>("downsample_ratio2")};
    // clip_bbox and scale_x_y is not used now!
    float nms_threshold = context.Attr<float>("nms_threshold");

    int batch = context.Input<framework::Tensor>("Boxes0")->dims()[0];
    TensorInfo* ts_info = new TensorInfo[batch * boxes_input.size()];
    for (int i = 0; i < batch * static_cast<int>(boxes_input.size()); i++) {
      hipMalloc(
          reinterpret_cast<void**>(&ts_info[i].bboxes_dev_ptr),
          ts_info[i].bbox_count_max_alloc * (5 + class_num) * sizeof(float));
      ts_info[i].bboxes_host_ptr = reinterpret_cast<float*>(malloc(
          ts_info[i].bbox_count_max_alloc * (5 + class_num) * sizeof(float)));
      hipMalloc(reinterpret_cast<void**>(&ts_info[i].bbox_count_device_ptr),
                 sizeof(int));
    }

    // box index counter in gpu memory
    // *bbox_index_device_ptr used by atomicAdd
    int* bbox_index_device_ptr;
    hipMalloc(reinterpret_cast<void**>(&bbox_index_device_ptr), sizeof(int));

    int total_bbox = 0;
    for (int batch_id = 0; batch_id < batch; batch_id++) {
      for (int input_id = 0; input_id < static_cast<int>(boxes_input.size());
           input_id++) {
        int c = boxes_input_dims[input_id][1];
        int h = boxes_input_dims[input_id][2];
        int w = boxes_input_dims[input_id][3];
        int ts_id = batch_id * static_cast<int>(boxes_input.size()) + input_id;
        int bbox_count_max_alloc = ts_info[ts_id].bbox_count_max_alloc;

        YoloTensorParseCuda(
            boxes_input[input_id] + batch_id * c * h * w,
            image_shape_data + batch_id * 2, image_scale_data + batch_id * 2,
            // output in gpu,must use 2-level pointer, because we may re-malloc
            &(ts_info[ts_id].bboxes_dev_ptr),
            &bbox_count_max_alloc,              // bbox_count_alloc_ptr boxes we
                                                // pre-allocate
            &(ts_info[ts_id].bbox_count_host),  // record bbox numbers
            ts_info[ts_id].bbox_count_device_ptr,  // for atomicAdd
            bbox_index_device_ptr,                 // for atomicAdd
            h, class_num, anchors_num[input_id], downsample_ratio[input_id] * h,
            downsample_ratio[input_id] * w, device_anchors_ptr[input_id],
            conf_thresh);

        // batch info update
        if (bbox_count_max_alloc > ts_info[ts_id].bbox_count_max_alloc) {
          ts_info[ts_id].bbox_count_max_alloc = bbox_count_max_alloc;
          ts_info[ts_id].bboxes_host_ptr = reinterpret_cast<float*>(
              realloc(ts_info[ts_id].bboxes_host_ptr,
                      bbox_count_max_alloc * (5 + class_num) * sizeof(float)));
        }
        // we need copy bbox_count_host boxes to cpu memory
        hipMemcpyAsync(
            ts_info[ts_id].bboxes_host_ptr, ts_info[ts_id].bboxes_dev_ptr,
            ts_info[ts_id].bbox_count_host * (5 + class_num) * sizeof(float),
            hipMemcpyDeviceToHost);
        total_bbox += ts_info[ts_id].bbox_count_host;
      }
    }

    boxes_scores_tensor->Resize({total_bbox > 0 ? total_bbox : 1, 6});
    float* boxes_scores_data =
        boxes_scores_tensor->mutable_data<float>(platform::CPUPlace());
    memset(boxes_scores_data, 0, sizeof(float) * 6);
    boxes_num_tensor->Resize({batch});
    int* boxes_num_data =
        boxes_num_tensor->mutable_data<int>(platform::CPUPlace());
    int boxes_scores_id = 0;

    // NMS
    for (int batch_id = 0; batch_id < batch; batch_id++) {
      std::vector<detection> bbox_det_vec;

      for (int input_id = 0; input_id < static_cast<int>(boxes_input.size());
           input_id++) {
        int ts_id = batch_id * static_cast<int>(boxes_input.size()) + input_id;
        int bbox_count = ts_info[ts_id].bbox_count_host;
        if (bbox_count <= 0) {
          continue;
        }
        float* bbox_host_ptr = ts_info[ts_id].bboxes_host_ptr;
        for (int bbox_index = 0; bbox_index < bbox_count; ++bbox_index) {
          detection bbox_det;
          memset(&bbox_det, 0, sizeof(detection));
          bbox_det.objectness = bbox_host_ptr[bbox_index * (5 + class_num) + 0];
          bbox_det.bbox.x = bbox_host_ptr[bbox_index * (5 + class_num) + 1];
          bbox_det.bbox.y = bbox_host_ptr[bbox_index * (5 + class_num) + 2];
          bbox_det.bbox.w =
              bbox_host_ptr[bbox_index * (5 + class_num) + 3] - bbox_det.bbox.x;
          bbox_det.bbox.h =
              bbox_host_ptr[bbox_index * (5 + class_num) + 4] - bbox_det.bbox.y;
          bbox_det.classes = class_num;
          bbox_det.prob =
              reinterpret_cast<float*>(malloc(class_num * sizeof(float)));
          int max_prob_class_id = -1;
          float max_class_prob = 0.0;
          for (int class_id = 0; class_id < class_num; class_id++) {
            float prob =
                bbox_host_ptr[bbox_index * (5 + class_num) + 5 + class_id];
            bbox_det.prob[class_id] = prob;
            if (prob > max_class_prob) {
              max_class_prob = prob;
              max_prob_class_id = class_id;
            }
          }
          bbox_det.max_prob_class_index = max_prob_class_id;
          bbox_det.sort_class = max_prob_class_id;
          bbox_det_vec.push_back(bbox_det);
        }
      }
      post_nms(&bbox_det_vec, nms_threshold, class_num);
      for (int i = 0; i < bbox_det_vec.size(); i++) {
        boxes_scores_data[boxes_scores_id++] =
            bbox_det_vec[i].max_prob_class_index;
        boxes_scores_data[boxes_scores_id++] = bbox_det_vec[i].objectness;
        boxes_scores_data[boxes_scores_id++] = bbox_det_vec[i].bbox.x;
        boxes_scores_data[boxes_scores_id++] = bbox_det_vec[i].bbox.y;
        boxes_scores_data[boxes_scores_id++] =
            bbox_det_vec[i].bbox.w + bbox_det_vec[i].bbox.x;
        boxes_scores_data[boxes_scores_id++] =
            bbox_det_vec[i].bbox.h + bbox_det_vec[i].bbox.y;
        free(bbox_det_vec[i].prob);
      }
      boxes_num_data[batch_id] = bbox_det_vec.size();
    }

    hipFree(bbox_index_device_ptr);
    for (int i = 0; i < batch * boxes_input.size(); i++) {
      hipFree(ts_info[i].bboxes_dev_ptr);
      hipFree(ts_info[i].bbox_count_device_ptr);
      free(ts_info[i].bboxes_host_ptr);
    }
    delete[] ts_info;
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(yolo_box_post, ops::YoloBoxPostKernel);
