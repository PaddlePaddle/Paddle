/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Copyright (c) 2022 NVIDIA Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fused/fused_gemm_epilogue_op.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/op_version_registry.h"
#include "paddle/fluid/framework/scope_guard.h"
#include "paddle/fluid/platform/bfloat16.h"
#include "paddle/fluid/platform/dynload/hipblaslt.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename DeviceContext, typename T>
class FusedGemmEpilogueKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();

    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* y = ctx.Input<Tensor>("Y");
    const Tensor* bias = ctx.Input<Tensor>("Bias");

    Tensor* out = ctx.Output<Tensor>("Out");
    Tensor* reserve_space = ctx.Output<Tensor>("ReserveSpace");

    bool trans_x = ctx.Attr<bool>("trans_x");
    bool trans_y = ctx.Attr<bool>("trans_y");

    std::string activation = ctx.Attr<std::string>("activation");
    VLOG(10) << "trans_x = " << trans_x << " , trans_y = " << trans_y
             << " , activation = " << activation;
    bool enable_auxiliary = reserve_space == nullptr ? false : true;

    dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));
    auto* out_data = out->data<T>();

    auto x_mat_dims =
        phi::flatten_to_2d(x->dims(), trans_x ? 1 : x->dims().size() - 1);
    // (M * K) * (K * N)
    int64_t M = trans_x ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = trans_y ? y->dims()[1] : y->dims()[0];
    int64_t N = trans_y ? y->dims()[0] : y->dims()[1];

    hipDataType mat_type = HIP_R_32F;
    hipDataType scale_type = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    if (std::is_same<T, paddle::platform::float16>::value) {
      mat_type = HIP_R_16F;
    }
    if (std::is_same<T, platform::bfloat16>::value) {
      mat_type = HIP_R_16BF;
    }
    if (std::is_same<T, double>::value) {
      mat_type = HIP_R_64F;
      scale_type = HIP_R_64F;
      compute_type = HIPBLAS_COMPUTE_64F;
    }

    hipblasLtMatmulDesc_t operation_desc = NULL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmulDescCreate(
        &operation_desc, compute_type, scale_type));
    hipblasOperation_t transx = trans_x ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transy = trans_y ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc,
            HIPBLASLT_MATMUL_DESC_TRANSB,
            &transx,
            sizeof(transx)));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc,
            HIPBLASLT_MATMUL_DESC_TRANSA,
            &transy,
            sizeof(transy)));

    hipblasLtEpilogue_t epiloque_func =
        get_epilogue_type_(activation, enable_auxiliary);
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc,
            HIPBLASLT_MATMUL_DESC_EPILOGUE,
            &epiloque_func,
            sizeof(epiloque_func)));
    const T* bias_data = bias->data<T>();
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc,
            HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
            &bias_data,
            sizeof(bias_data)));

    if (enable_auxiliary && activation != "none") {
      // Note (Ming Huang): The initialization of ReseveSpace is happened in the
      // dev_ctx.Alloc. Therefore, we set real date type up here.
      if (activation == "relu") {
        paddle::experimental::DataType rs_type =
            paddle::experimental::DataType::BOOL;
        size_t reserve_space_size =
            phi::product(reserve_space->dims()) * SizeOf(rs_type);
        dev_ctx.Alloc(reserve_space, rs_type, reserve_space_size);
      } else {
        size_t reserve_space_size =
            phi::product(reserve_space->dims()) * sizeof(T);
        dev_ctx.Alloc<T>(reserve_space, reserve_space_size);
      }

      void* aux_data = reserve_space->data();

      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              operation_desc,
              HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
              &aux_data,
              sizeof(aux_data)));
      int64_t aux_ld = N;
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              operation_desc,
              HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
              &aux_ld,
              sizeof(aux_ld)));
    }

    hipblasLtMatrixLayout_t x_desc = NULL, y_desc = NULL, out_desc = NULL;
    if (trans_x)
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &x_desc, mat_type, M, K, M));
    else
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &x_desc, mat_type, K, M, K));
    if (trans_y)
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &y_desc, mat_type, K, N, K));
    else
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &y_desc, mat_type, N, K, N));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &out_desc, mat_type, N, M, N));

    hipblasLtHandle_t lt_handle = dev_ctx.cublaslt_handle();
    // NOTE(zengjinle): I do not know whether the 4MB workspace size is
    // "enough". I just followed the settings from the NVIDIA MLPerf BERT code.
    size_t workspace_size = static_cast<size_t>(4) * 1024 * 1024;
    hipStream_t stream = dev_ctx.stream();
    memory::allocation::AllocationPtr workspace = memory::Alloc(
        dev_ctx.GetPlace(),
        workspace_size,
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

    double alpha64 = 1.0, beta64 = 0.0;
    float alpha32 = 1.0f, beta32 = 0.0f;
    void *alpha = nullptr, *beta = nullptr;
    if (std::is_same<T, double>::value) {
      alpha = &alpha64;
      beta = &beta64;
    } else {
      alpha = &alpha32;
      beta = &beta32;
    }

    const auto* y_data = y->data<T>();
    const auto* x_data = x->data<T>();

    auto algo = GemmEpilogueAlgoCache::Instance().GetGemmAlgo(lt_handle,
                                                              operation_desc,
                                                              y_desc,
                                                              x_desc,
                                                              out_desc,
                                                              alpha,
                                                              beta,
                                                              y_data,
                                                              x_data,
                                                              out_data,
                                                              stream,
                                                              workspace->ptr(),
                                                              workspace_size);
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmul(lt_handle,
                                          operation_desc,
                                          alpha,
                                          y_data,
                                          y_desc,
                                          x_data,
                                          x_desc,
                                          beta,
                                          out_data,
                                          out_desc,
                                          out_data,
                                          out_desc,
                                          algo,
                                          workspace->ptr(),
                                          workspace_size,
                                          stream));

    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescDestroy(operation_desc));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatrixLayoutDestroy(y_desc));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatrixLayoutDestroy(x_desc));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatrixLayoutDestroy(out_desc));
  }

 private:
  static hipblasLtEpilogue_t get_epilogue_type_(const std::string& activation,
                                               bool enable_auxiliary) {
    if (activation == "relu") {
      return enable_auxiliary ? HIPBLASLT_EPILOGUE_RELU_AUX_BIAS
                              : HIPBLASLT_EPILOGUE_RELU_BIAS;
    } else if (activation == "gelu") {
      return enable_auxiliary ? HIPBLASLT_EPILOGUE_GELU_AUX_BIAS
                              : HIPBLASLT_EPILOGUE_GELU_BIAS;
    } else if (activation == "none") {
      return HIPBLASLT_EPILOGUE_BIAS;
    } else {
      PADDLE_ENFORCE_EQ(
          true,
          false,
          platform::errors::InvalidArgument(
              "The activation attribute of fused_gemm_epilogue op should be"
              " one of {\"none\", \"relu\", \"gelu\"}. But received %s."
              "But received activation=%s.",
              activation));
    }
  }
};

enum FusedGEMMGradInType { kDX = 0, kDY = 1, kDZ = 2 };

template <bool TransX, bool TransY>
struct FusedGEMMGradTrait;

template <>
struct FusedGEMMGradTrait<false, false> {
  static constexpr auto kXGradA = FusedGEMMGradInType::kDZ;
  static constexpr auto kXGradB = FusedGEMMGradInType::kDY;
  static constexpr auto kXGradATrans = false;
  static constexpr auto kXGradBTrans = true;

  static constexpr auto kYGradA = FusedGEMMGradInType::kDX;
  static constexpr auto kYGradB = FusedGEMMGradInType::kDZ;
  static constexpr auto kYGradATrans = true;
  static constexpr auto kYGradBTrans = false;
};

template <>
struct FusedGEMMGradTrait<true, false> {
  static constexpr auto kXGradA = FusedGEMMGradInType::kDY;
  static constexpr auto kXGradB = FusedGEMMGradInType::kDZ;
  static constexpr auto kXGradATrans = false;
  static constexpr auto kXGradBTrans = true;

  static constexpr auto kYGradA = FusedGEMMGradInType::kDX;
  static constexpr auto kYGradB = FusedGEMMGradInType::kDZ;
  static constexpr auto kYGradATrans = false;
  static constexpr auto kYGradBTrans = false;
};

template <>
struct FusedGEMMGradTrait<false, true> {
  static constexpr auto kXGradA = FusedGEMMGradInType::kDZ;
  static constexpr auto kXGradB = FusedGEMMGradInType::kDY;
  static constexpr auto kXGradATrans = false;
  static constexpr auto kXGradBTrans = false;

  static constexpr auto kYGradA = FusedGEMMGradInType::kDZ;
  static constexpr auto kYGradB = FusedGEMMGradInType::kDX;
  static constexpr auto kYGradATrans = true;
  static constexpr auto kYGradBTrans = false;
};

template <>
struct FusedGEMMGradTrait<true, true> {
  static constexpr auto kXGradA = FusedGEMMGradInType::kDY;
  static constexpr auto kXGradB = FusedGEMMGradInType::kDZ;
  static constexpr auto kXGradATrans = true;
  static constexpr auto kXGradBTrans = true;

  static constexpr auto kYGradA = FusedGEMMGradInType::kDZ;
  static constexpr auto kYGradB = FusedGEMMGradInType::kDX;
  static constexpr auto kYGradATrans = true;
  static constexpr auto kYGradBTrans = true;
};

static constexpr auto BoolToCuBlasEnum(bool transpose) {
  return transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
}

template <typename DeviceContext, typename T>
class FusedGemmEpilogueGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    bool transpose_x = ctx.Attr<bool>("trans_x");
    bool transpose_y = ctx.Attr<bool>("trans_y");

    if (transpose_x) {
      if (transpose_y) {
        ComputeImpl<true, true>(ctx);
      } else {
        ComputeImpl<true, false>(ctx);
      }
    } else {
      if (transpose_y) {
        ComputeImpl<false, true>(ctx);
      } else {
        ComputeImpl<false, false>(ctx);
      }
    }
  }

 private:
  template <bool TransX, bool TransY>
  static void ComputeImpl(const framework::ExecutionContext& ctx) {
    using Trait = FusedGEMMGradTrait<TransX, TransY>;
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();
    const Tensor* dout = ctx.Input<Tensor>("DOut");
    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* y = ctx.Input<Tensor>("Y");
    const Tensor* reserve_space = ctx.Input<Tensor>("ReserveSpace");

    Tensor* dx = ctx.Output<Tensor>("DX");
    Tensor* dy = ctx.Output<Tensor>("DY");
    Tensor* dbias = ctx.Output<Tensor>("DBias");

    std::string activation_grad = ctx.Attr<std::string>("activation_grad");

    VLOG(10) << "trans_x = " << TransX << " , trans_y = " << TransY
             << " , activation_grad = " << activation_grad;

    auto x_mat_dims =
        phi::flatten_to_2d(x->dims(), TransX ? 1 : x->dims().size() - 1);

    // (M * K) * (K * N)
    int64_t M = TransX ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = TransY ? y->dims()[1] : y->dims()[0];
    int64_t N = TransY ? y->dims()[0] : y->dims()[1];

    VLOG(10) << "M = " << M << " , K = " << K << " , N = " << N;

    hipDataType mat_type = HIP_R_32F;
    hipDataType scale_type = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    if (std::is_same<T, paddle::platform::float16>::value) {
      mat_type = HIP_R_16F;
    }
    if (std::is_same<T, platform::bfloat16>::value) {
      mat_type = HIP_R_16BF;
    }
    if (std::is_same<T, double>::value) {
      mat_type = HIP_R_64F;
      scale_type = HIP_R_64F;
      compute_type = HIPBLAS_COMPUTE_64F;
    }

    hipblasLtHandle_t lt_handle = dev_ctx.cublaslt_handle();
    // NOTE(zengjinle): I do not know whether the 4MB workspace size is
    // "enough". I just followed the settings from the NVIDIA MLPerf BERT code.
    size_t workspace_size = static_cast<size_t>(4) * 1024 * 1024;
    const hipblasLtMatmulAlgo_t* algo = nullptr;
    hipStream_t stream = dev_ctx.stream();

    double alpha64 = 1.0, beta64 = 0.0;
    float alpha32 = 1.0f, beta32 = 0.0f;
    void *alpha = nullptr, *beta = nullptr;
    if (std::is_same<T, double>::value) {
      alpha = &alpha64;
      beta = &beta64;
    } else {
      alpha = &alpha32;
      beta = &beta32;
    }

    hipblasLtMatrixLayout_t dout_desc = nullptr, dout_trans_desc = nullptr;
    hipblasLtMatrixLayout_t x_desc = nullptr, x_trans_desc = nullptr;
    hipblasLtMatrixLayout_t y_desc = nullptr, y_trans_desc = nullptr;
    hipblasLtMatrixLayout_t dx_desc = nullptr, dy_desc = nullptr;
    hipblasLtMatmulDesc_t dx_operation_desc = nullptr,
                         dy_operation_desc = nullptr;

    DEFINE_PADDLE_SCOPE_GUARD([&] {
      auto descs = {dout_desc,
                    dout_trans_desc,
                    x_desc,
                    x_trans_desc,
                    y_desc,
                    y_trans_desc,
                    dx_desc,
                    dy_desc};
      for (auto desc : descs) {
        if (desc) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              platform::dynload::hipblasLtMatrixLayoutDestroy(desc));
        }
      }

      if (dx_operation_desc) {
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatmulDescDestroy(dx_operation_desc));
      }

      if (dy_operation_desc) {
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatmulDescDestroy(dy_operation_desc));
      }
    });

    auto x_row = TransX ? K : M;
    auto x_col = TransX ? M : K;
    auto y_row = TransY ? N : K;
    auto y_col = TransY ? K : N;
    auto z_row = TransX ? N : M;
    auto z_col = TransX ? M : N;

    // dx = func(dout, y)
    if (dx) {
      constexpr auto kXGradAIsDZ = (Trait::kXGradA == FusedGEMMGradInType::kDZ);
      hipblasLtMatrixLayout_t *dx_dout_desc, *dx_y_desc;

      if (TransX) {
        dx_dout_desc = &dout_trans_desc;
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatrixLayoutCreate(
                dx_dout_desc, mat_type, z_row, z_col, z_row));
      } else {
        dx_dout_desc = &dout_desc;
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatrixLayoutCreate(
                dx_dout_desc, mat_type, z_col, z_row, z_col));
      }

      dx_y_desc = &y_trans_desc;
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          dx_y_desc, mat_type, y_col, y_row, y_col));

      auto& a_desc = kXGradAIsDZ ? (*dx_dout_desc) : (*dx_y_desc);
      auto& b_desc = kXGradAIsDZ ? (*dx_y_desc) : (*dx_dout_desc);
      auto a_trans = BoolToCuBlasEnum(Trait::kXGradATrans);
      auto b_trans = BoolToCuBlasEnum(Trait::kXGradBTrans);

      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &dx_desc, mat_type, x_col, x_row, x_col));

      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmulDescCreate(
          &dx_operation_desc, compute_type, scale_type));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dx_operation_desc,
              HIPBLASLT_MATMUL_DESC_TRANSB,
              &a_trans,
              sizeof(a_trans)));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dx_operation_desc,
              HIPBLASLT_MATMUL_DESC_TRANSA,
              &b_trans,
              sizeof(b_trans)));

      hipblasLtEpilogue_t epiloque_func_for_dx =
          get_epilogue_type_(activation_grad);
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dx_operation_desc,
              HIPBLASLT_MATMUL_DESC_EPILOGUE,
              &epiloque_func_for_dx,
              sizeof(epiloque_func_for_dx)));

      if (activation_grad != "none") {
        auto* aux_data = reserve_space->data();
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatmulDescSetAttribute(
                dx_operation_desc,
                HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
                &aux_data,
                sizeof(aux_data)));
        int64_t aux_ld = TransX ? M : K;
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatmulDescSetAttribute(
                dx_operation_desc,
                HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD,
                &aux_ld,
                sizeof(aux_ld)));
      }

      auto dx_workspace = memory::Alloc(
          dev_ctx.GetPlace(),
          workspace_size,
          phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

      auto* dx_data = dev_ctx.Alloc<T>(dx, dx->numel() * sizeof(T));
      const auto* y_data = y->data<T>();
      const auto* dout_data = dout->data<T>();
      const auto* a_data = kXGradAIsDZ ? dout_data : y_data;
      const auto* b_data = kXGradAIsDZ ? y_data : dout_data;

      auto algo =
          GemmEpilogueAlgoCache::Instance().GetGemmAlgo(lt_handle,
                                                        dx_operation_desc,
                                                        b_desc,
                                                        a_desc,
                                                        dx_desc,
                                                        alpha,
                                                        beta,
                                                        b_data,
                                                        a_data,
                                                        dx_data,
                                                        stream,
                                                        dx_workspace->ptr(),
                                                        workspace_size);

      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmul(lt_handle,
                                            dx_operation_desc,
                                            alpha,
                                            b_data,
                                            b_desc,
                                            a_data,
                                            a_desc,
                                            beta,
                                            dx_data,
                                            dx_desc,
                                            dx_data,
                                            dx_desc,
                                            algo,
                                            dx_workspace->ptr(),
                                            workspace_size,
                                            stream));
    }

    // dy = func(dout, x)
    if (dy) {
      constexpr auto kYGradAIsDZ = (Trait::kYGradA == FusedGEMMGradInType::kDZ);

      hipblasLtMatrixLayout_t *dy_dout_desc = nullptr, *dy_x_desc = nullptr;
      if (TransX) {
        dy_dout_desc = &dout_trans_desc;
        if (dout_trans_desc == nullptr) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              platform::dynload::hipblasLtMatrixLayoutCreate(
                  dy_dout_desc, mat_type, z_row, z_col, z_row));
        }
      } else {
        dy_dout_desc = &dout_desc;
        if (dout_desc == nullptr) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              platform::dynload::hipblasLtMatrixLayoutCreate(
                  dy_dout_desc, mat_type, z_col, z_row, z_col));
        }
      }

      dy_x_desc = &x_trans_desc;
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          dy_x_desc, mat_type, x_col, x_row, x_col));

      auto& a_desc = kYGradAIsDZ ? (*dy_dout_desc) : (*dy_x_desc);
      auto& b_desc = kYGradAIsDZ ? (*dy_x_desc) : (*dy_dout_desc);
      auto a_trans = BoolToCuBlasEnum(Trait::kYGradATrans);
      auto b_trans = BoolToCuBlasEnum(Trait::kYGradBTrans);

      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &dy_desc, mat_type, y_col, y_row, y_col));

      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmulDescCreate(
          &dy_operation_desc, compute_type, scale_type));

      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dy_operation_desc,
              HIPBLASLT_MATMUL_DESC_TRANSB,
              &a_trans,
              sizeof(a_trans)));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dy_operation_desc,
              HIPBLASLT_MATMUL_DESC_TRANSA,
              &b_trans,
              sizeof(b_trans)));

      hipblasLtEpilogue_t epiloque_func_for_dy;
      if (dbias == nullptr) {
        epiloque_func_for_dy = HIPBLASLT_EPILOGUE_DEFAULT;
      } else {
        if (TransY) {
          epiloque_func_for_dy = HIPBLASLT_EPILOGUE_BGRADB;
        } else {
          epiloque_func_for_dy = HIPBLASLT_EPILOGUE_BGRADA;
        }
      }

      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dy_operation_desc,
              HIPBLASLT_MATMUL_DESC_EPILOGUE,
              &epiloque_func_for_dy,
              sizeof(epiloque_func_for_dy)));

      if (dbias) {
        auto* dbias_data = dev_ctx.Alloc<T>(dbias, dbias->numel() * sizeof(T));
        PADDLE_ENFORCE_GPU_SUCCESS(
            platform::dynload::hipblasLtMatmulDescSetAttribute(
                dy_operation_desc,
                HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                &dbias_data,
                sizeof(dbias_data)));
      }

      auto dy_workspace = memory::Alloc(
          dev_ctx.GetPlace(),
          workspace_size,
          phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
      auto* dy_data = dev_ctx.Alloc<T>(dy, dy->numel() * sizeof(T));
      const auto* dout_data = dout->data<T>();
      const auto* x_data = x->data<T>();
      const auto* a_data = kYGradAIsDZ ? dout_data : x_data;
      const auto* b_data = kYGradAIsDZ ? x_data : dout_data;

      auto algo =
          GemmEpilogueAlgoCache::Instance().GetGemmAlgo(lt_handle,
                                                        dy_operation_desc,
                                                        b_desc,
                                                        a_desc,
                                                        dy_desc,
                                                        alpha,
                                                        beta,
                                                        b_data,
                                                        a_data,
                                                        dy_data,
                                                        stream,
                                                        dy_workspace->ptr(),
                                                        workspace_size);

      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmul(lt_handle,
                                            dy_operation_desc,
                                            alpha,
                                            b_data,
                                            b_desc,
                                            a_data,
                                            a_desc,
                                            beta,
                                            dy_data,
                                            dy_desc,
                                            dy_data,
                                            dy_desc,
                                            algo,
                                            dy_workspace->ptr(),
                                            workspace_size,
                                            stream));
    }
  }

 private:
  static hipblasLtEpilogue_t get_epilogue_type_(
      const std::string& activation_grad) {
    if (activation_grad == "relu_grad") {
      return CUBLASLT_EPILOGUE_DRELU;
    } else if (activation_grad == "gelu_grad") {
      return HIPBLASLT_EPILOGUE_DGELU;
    } else if (activation_grad == "none") {
      return HIPBLASLT_EPILOGUE_DEFAULT;
    } else {
      PADDLE_ENFORCE_EQ(
          true,
          false,
          platform::errors::InvalidArgument(
              "The activation_grad attribute of fused_gemm_epilogue op should "
              "be"
              " one of {\"none\", \"relu\", \"gelu\"}. But received %s."
              "But received activation_grad=%s.",
              activation_grad));
    }
  }
};

}  // namespace operators
}  // namespace paddle

#if CUDA_VERSION >= 11060
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fused_gemm_epilogue,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, float>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, double>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, paddle::platform::float16>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, paddle::platform::bfloat16>);

REGISTER_OP_CUDA_KERNEL(
    fused_gemm_epilogue_grad,
    ops::FusedGemmEpilogueGradKernel<phi::GPUContext, float>,
    ops::FusedGemmEpilogueGradKernel<phi::GPUContext, double>,
    ops::FusedGemmEpilogueGradKernel<phi::GPUContext,
                                     paddle::platform::float16>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, paddle::platform::bfloat16>);
#endif
