/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Copyright (c) 2022 NVIDIA Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fused/fused_gemm_epilogue_op.h"
#include "paddle/fluid/platform/dynload/hipblaslt.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename DeviceContext, typename T>
class FusedGemmEpilogueKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* y = ctx.Input<Tensor>("Y");
    const Tensor* bias = ctx.Input<Tensor>("bias");

    Tensor* out = ctx.Output<Tensor>("out");

    bool trans_x = ctx.Attr<bool>("trans_x");
    bool trans_y = ctx.Attr<bool>("trans_y");

    std::string activation = ctx.Attr<std::string>("activation");
    std::string auxiliary_key = ctx.Attr<std::string>("auxiliary_key");
    bool enable_auxiliary = auxiliary_key.size() <= 0 ? false : true;

    out->mutable_data<T>(ctx.GetPlace());
    auto* out_data = out->data<T>();

    auto x_mat_dims =
        framework::flatten_to_2d(x->dims(), trans_x ? 1 : x->dims().size() - 1);
    int64_t M = trans_x ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = trans_y ? y->dims()[1] : y->dims()[0];
    int64_t N = trans_y ? y->dims()[0] : y->dims()[1];

    hipDataType mat_type = HIP_R_32F;
    hipDataType scale_type = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    if (std::is_same<T, paddle::platform::float16>::value) {
      mat_type = HIP_R_16F;
      scale_type = HIP_R_16F;
    }
    if (std::is_same<T, double>::value) {
      mat_type = HIP_R_64F;
      scale_type = HIP_R_64F;
      compute_type = HIPBLAS_COMPUTE_64F;
    }

    hipblasLtMatmulDesc_t operation_desc = NULL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmulDescCreate(
        &operation_desc, compute_type, scale_type));
    hipblasOperation_t transx = trans_x ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transy = trans_y ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &transx,
            sizeof(transx)));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &transy,
            sizeof(transy)));

    hipblasLtEpilogue_t epiloque_func =
        get_epilogue_type_(activation, enable_auxiliary);
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epiloque_func,
            sizeof(epiloque_func)));
    const T* bias_data = bias->data<T>();
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            operation_desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias_data,
            sizeof(bias_data)));
    if (enable_auxiliary && activation != "none") {
      if (EpilogueSingleton::Instance().Data(auxiliary_key).auxiliary ==
          nullptr) {
        size_t unit_size = activation == "relu" ? 2 : sizeof(T);
        size_t auxiliary_size =
            static_cast<size_t>(framework::product(out->dims())) * unit_size;
        EpilogueSingleton::Instance().Data(auxiliary_key).auxiliary =
            memory::Alloc(dev_ctx, auxiliary_size);
      }

      auto* aux_data =
          EpilogueSingleton::Instance().Data(auxiliary_key).auxiliary->ptr();
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
              &aux_data, sizeof(aux_data)));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &N,
              sizeof(N)));
    }

    hipblasLtMatrixLayout_t x_desc = NULL, y_desc = NULL, out_desc = NULL;
    if (trans_x)
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &x_desc, mat_type, M, K, M));
    else
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &x_desc, mat_type, K, M, K));
    if (trans_y)
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &y_desc, mat_type, K, N, K));
    else
      PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
          &y_desc, mat_type, N, K, N));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &out_desc, mat_type, N, M, N));

    hipblasLtHandle_t lt_handle = dev_ctx.cublaslt_handle();
    size_t workspace_size = 4 * 1024 * 1024;
    const hipblasLtMatmulAlgo_t* algo = nullptr;
    hipStream_t stream = 0;
    memory::allocation::AllocationPtr workspace =
        memory::Alloc(dev_ctx, workspace_size);

    double alpha64 = 1.0, beta64 = 0.0;
    float alpha32 = 1.0f, beta32 = 0.0f;
    void *alpha, *beta;
    if (std::is_same<T, double>::value) {
      alpha = &alpha64;
      beta = &beta64;
    } else {
      alpha = &alpha32;
      beta = &beta32;
    }

    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmul(
        lt_handle, operation_desc, alpha, y->data<T>(), y_desc, x->data<T>(),
        x_desc, beta, out_data, out_desc, out_data, out_desc, algo,
        workspace->ptr(), workspace_size, stream));
  }

 private:
  static hipblasLtEpilogue_t get_epilogue_type_(std::string activation,
                                               bool enable_auxiliary) {
    if (activation == "relu") {
      return enable_auxiliary ? HIPBLASLT_EPILOGUE_RELU_AUX_BIAS
                              : HIPBLASLT_EPILOGUE_RELU_BIAS;
    } else if (activation == "gelu") {
      return enable_auxiliary ? HIPBLASLT_EPILOGUE_GELU_AUX_BIAS
                              : HIPBLASLT_EPILOGUE_GELU_BIAS;
    } else if (activation == "none") {
      return HIPBLASLT_EPILOGUE_BIAS;
    } else {
      PADDLE_ENFORCE_EQ(
          true, false,
          platform::errors::InvalidArgument(
              "The activation attribute of fused_gemm_epilogue op should be"
              " one of {\"none\", \"relu\", \"gelu\"}. But received %s."
              "But received activation=%s.",
              activation));
    }
  }
};

template <typename DeviceContext, typename T>
class FusedGemmEpilogueGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    const Tensor* dout = ctx.Input<Tensor>("DOut");
    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* y = ctx.Input<Tensor>("Y");

    Tensor* dx = ctx.Output<Tensor>("DX");
    Tensor* dy = ctx.Output<Tensor>("DY");
    Tensor* dbias = ctx.Output<Tensor>("DBias");

    std::string activation_grad = ctx.Attr<std::string>("activation_grad");
    std::string auxiliary_key = ctx.Attr<std::string>("auxiliary_key");

    auto dout_mat_dims =
        framework::flatten_to_2d(dout->dims(), dout->dims().size() - 1);
    auto x_mat_dims = framework::flatten_to_2d(x->dims(), x->dims().size() - 1);

    int64_t M = x_mat_dims[0];
    int64_t K = y->dims()[0];
    int64_t N = y->dims()[1];

    hipDataType mat_type = HIP_R_32F;
    hipDataType scale_type = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;
    if (std::is_same<T, paddle::platform::float16>::value) {
      mat_type = HIP_R_16F;
      scale_type = HIP_R_16F;
    }
    if (std::is_same<T, double>::value) {
      mat_type = HIP_R_64F;
      scale_type = HIP_R_64F;
      compute_type = HIPBLAS_COMPUTE_64F;
    }

    hipblasLtHandle_t lt_handle = dev_ctx.cublaslt_handle();
    size_t workspace_size = 4 * 1024 * 1024;
    const hipblasLtMatmulAlgo_t* algo = nullptr;
    hipStream_t stream = 0;

    double alpha64 = 1.0, beta64 = 0.0;
    float alpha32 = 1.0f, beta32 = 0.0f;
    void *alpha, *beta;
    if (std::is_same<T, double>::value) {
      alpha = &alpha64;
      beta = &beta64;
    } else {
      alpha = &alpha32;
      beta = &beta32;
    }

    hipblasLtMatmulDesc_t dx_operation_desc = NULL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmulDescCreate(
        &dx_operation_desc, compute_type, scale_type));
    hipblasOperation_t trans_dout = HIPBLAS_OP_N;
    hipblasOperation_t trans_y = HIPBLAS_OP_T;
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dx_operation_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &trans_dout,
            sizeof(trans_dout)));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dx_operation_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans_y,
            sizeof(trans_y)));
    hipblasLtEpilogue_t epiloque_func_for_dx =
        get_epilogue_type_(activation_grad);
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dx_operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE,
            &epiloque_func_for_dx, sizeof(epiloque_func_for_dx)));

    if (activation_grad != "none") {
      auto* aux_data =
          EpilogueSingleton::Instance().Data(auxiliary_key).auxiliary->ptr();
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dx_operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER,
              &aux_data, sizeof(aux_data)));
      PADDLE_ENFORCE_GPU_SUCCESS(
          platform::dynload::hipblasLtMatmulDescSetAttribute(
              dx_operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &N,
              sizeof(N)));
    }

    hipblasLtMatrixLayout_t dout_desc = NULL, y_desc = NULL, dx_desc = NULL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &dout_desc, mat_type, N, M, N));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &y_desc, mat_type, N, K, N));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &dx_desc, mat_type, K, M, K));

    memory::allocation::AllocationPtr dx_workspace =
        memory::Alloc(dev_ctx, workspace_size);

    dx->mutable_data<T>(ctx.GetPlace());
    auto* dx_data = dx->data<T>();
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmul(
        lt_handle, dx_operation_desc, alpha, y->data<T>(), y_desc,
        dout->data<T>(), dout_desc, beta, dx_data, dx_desc, dx_data, dx_desc,
        algo, dx_workspace->ptr(), workspace_size, stream));

    hipblasLtMatmulDesc_t dy_operation_desc = NULL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmulDescCreate(
        &dy_operation_desc, compute_type, scale_type));
    hipblasOperation_t trans_x = HIPBLAS_OP_T;
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dy_operation_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans_dout,
            sizeof(trans_dout)));
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dy_operation_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &trans_x,
            sizeof(trans_x)));
    hipblasLtEpilogue_t epiloque_func_for_dy = HIPBLASLT_EPILOGUE_BGRADA;
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dy_operation_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE,
            &epiloque_func_for_dy, sizeof(epiloque_func_for_dy)));

    dbias->mutable_data<T>(ctx.GetPlace());
    auto* dbias_data = dbias->data<T>();
    PADDLE_ENFORCE_GPU_SUCCESS(
        platform::dynload::hipblasLtMatmulDescSetAttribute(
            dy_operation_desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &dbias_data,
            sizeof(dbias_data)));

    hipblasLtMatrixLayout_t x_desc = NULL, dy_desc = NULL;
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &x_desc, mat_type, K, M, K));
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatrixLayoutCreate(
        &dy_desc, mat_type, N, K, N));

    memory::allocation::AllocationPtr dy_workspace =
        memory::Alloc(dev_ctx, workspace_size);

    dy->mutable_data<T>(ctx.GetPlace());
    auto* dy_data = dy->data<T>();
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipblasLtMatmul(
        lt_handle, dy_operation_desc, alpha, dout->data<T>(), dout_desc,
        x->data<T>(), x_desc, beta, dy_data, dy_desc, dy_data, dy_desc, algo,
        dy_workspace->ptr(), workspace_size, stream));
  }

 private:
  static hipblasLtEpilogue_t get_epilogue_type_(std::string activation_grad) {
    if (activation_grad == "relu_grad") {
      return CUBLASLT_EPILOGUE_DRELU;
    } else if (activation_grad == "gelu_grad") {
      return HIPBLASLT_EPILOGUE_DGELU;
    } else if (activation_grad == "none") {
      return HIPBLASLT_EPILOGUE_DEFAULT;
    } else {
      PADDLE_ENFORCE_EQ(
          true, false,
          platform::errors::InvalidArgument(
              "The activation_grad attribute of fused_gemm_epilogue op should "
              "be"
              " one of {\"none\", \"relu\", \"gelu\"}. But received %s."
              "But received activation_grad=%s.",
              activation_grad));
    }
  }
};

}  // namespace operators
}  // namespace paddle

#if CUDA_VERSION >= 11060
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fused_gemm_epilogue,
    ops::FusedGemmEpilogueKernel<paddle::platform::CUDADeviceContext, float>,
    ops::FusedGemmEpilogueKernel<paddle::platform::CUDADeviceContext, double>,
    ops::FusedGemmEpilogueKernel<paddle::platform::CUDADeviceContext,
                                 paddle::platform::float16>);

REGISTER_OP_CUDA_KERNEL(
    fused_gemm_epilogue_grad,
    ops::FusedGemmEpilogueGradKernel<paddle::platform::CUDADeviceContext,
                                     float>,
    ops::FusedGemmEpilogueGradKernel<paddle::platform::CUDADeviceContext,
                                     double>,
    ops::FusedGemmEpilogueGradKernel<paddle::platform::CUDADeviceContext,
                                     paddle::platform::float16>);
#endif
