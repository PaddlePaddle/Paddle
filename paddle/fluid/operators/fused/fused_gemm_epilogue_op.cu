/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Copyright (c) 2022 NVIDIA Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/fused/fused_gemm_epilogue_op.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/op_version_registry.h"
#include "paddle/fluid/platform/bfloat16.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

#if CUDA_VERSION >= 11060

template <typename DeviceContext, typename T>
class FusedGemmEpilogueKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();

    const phi::DenseTensor* x = ctx.Input<phi::DenseTensor>("X");
    const phi::DenseTensor* y = ctx.Input<phi::DenseTensor>("Y");
    const phi::DenseTensor* bias = ctx.Input<phi::DenseTensor>("Bias");

    phi::DenseTensor* out = ctx.Output<phi::DenseTensor>("Out");
    phi::DenseTensor* reserve_space =
        ctx.Output<phi::DenseTensor>("ReserveSpace");

    bool trans_x = ctx.Attr<bool>("trans_x");
    bool trans_y = ctx.Attr<bool>("trans_y");

    std::string activation = ctx.Attr<std::string>("activation");
    dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));

    // (M * K) * (K * N)
    auto x_mat_dims =
        phi::flatten_to_2d(x->dims(), trans_x ? 1 : x->dims().size() - 1);
    int64_t M = trans_x ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = trans_y ? y->dims()[1] : y->dims()[0];
    int64_t N = trans_y ? y->dims()[0] : y->dims()[1];

    ComputeFusedGemmEpilogueForward<T>(dev_ctx,
                                       x,
                                       y,
                                       bias,
                                       M,
                                       N,
                                       K,
                                       trans_x,
                                       trans_y,
                                       activation,
                                       out,
                                       reserve_space);
  }
};

template <typename DeviceContext, typename T>
class FusedGemmEpilogueGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();

    const phi::DenseTensor* dout = ctx.Input<phi::DenseTensor>("DOut");
    const phi::DenseTensor* x = ctx.Input<phi::DenseTensor>("X");
    const phi::DenseTensor* y = ctx.Input<phi::DenseTensor>("Y");
    const phi::DenseTensor* reserve_space =
        ctx.Input<phi::DenseTensor>("ReserveSpace");

    phi::DenseTensor* dx = ctx.Output<phi::DenseTensor>("DX");
    phi::DenseTensor* dy = ctx.Output<phi::DenseTensor>("DY");
    phi::DenseTensor* dbias = ctx.Output<phi::DenseTensor>("DBias");

    std::string activation_grad = ctx.Attr<std::string>("activation_grad");

    bool trans_x = ctx.Attr<bool>("trans_x");
    bool trans_y = ctx.Attr<bool>("trans_y");

    // (M * K) * (K * N)
    auto x_mat_dims =
        phi::flatten_to_2d(x->dims(), trans_x ? 1 : x->dims().size() - 1);
    int64_t M = trans_x ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = trans_y ? y->dims()[1] : y->dims()[0];
    int64_t N = trans_y ? y->dims()[0] : y->dims()[1];

    ComputeFusedGemmEpilogueBackward<T>(dev_ctx,
                                        dout,
                                        x,
                                        y,
                                        reserve_space,
                                        M,
                                        N,
                                        K,
                                        trans_x,
                                        trans_y,
                                        activation_grad,
                                        dx,
                                        dy,
                                        dbias);
  }
};
#endif

}  // namespace operators
}  // namespace paddle

#if CUDA_VERSION >= 11060
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    fused_gemm_epilogue,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, float>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, double>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, paddle::platform::float16>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, paddle::platform::bfloat16>);

REGISTER_OP_CUDA_KERNEL(
    fused_gemm_epilogue_grad,
    ops::FusedGemmEpilogueGradKernel<phi::GPUContext, float>,
    ops::FusedGemmEpilogueGradKernel<phi::GPUContext, double>,
    ops::FusedGemmEpilogueGradKernel<phi::GPUContext,
                                     paddle::platform::float16>,
    ops::FusedGemmEpilogueKernel<phi::GPUContext, paddle::platform::bfloat16>);
#endif
