#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/conv_cudnn_op_cache.h"
#include "paddle/fluid/platform/cudnn_helper.h"

DECLARE_uint64(conv_workspace_size_limit);

namespace paddle {
namespace operators {

#if CUDNN_VERSION >= 7100
using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using ScopedActivationDescriptor = platform::ScopedActivationDescriptor;
using DataLayout = platform::DataLayout;

using ScopedPoolingDescriptor = platform::ScopedPoolingDescriptor;
using PoolingMode = platform::PoolingMode;
template <typename T>
using ScalingParamType = typename platform::CudnnDataType<T>::ScalingParamType;

template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;

template <typename T>
class CUDNNConvInceptionFusionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto* input = ctx.Input<Tensor>("Input");
    auto filters = ctx.MultiInput<framework::Tensor>("Filter");
    auto bias = ctx.MultiInput<framework::Tensor>("Bias");

    auto* output = ctx.Output<Tensor>("Output");
    auto temp_outs = ctx.MultiOutput<framework::Tensor>("TempOutput");

    const std::string pool_type = ctx.Attr<std::string>("pooling_type");
    const std::string activation = ctx.Attr<std::string>("activation");
    const bool exclusive = ctx.Attr<bool>("exclusive");

    int64_t user_workspace_size =
        static_cast<size_t>(ctx.Attr<int>("workspace_size_MB"));

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    T* temp_data = temp_outs[0]->mutable_data<T>(input->dims(), ctx.GetPlace());

    DataLayout layout = DataLayout::kNCHW;
    std::vector<int> in_dim = framework::vectorize<int>(input->dims());

    // ------------------- cudnn descriptors ---------------------
    PoolingMode pooling_mode;
    if (pool_type == "max") {
      pooling_mode = PoolingMode::kMaximum;
    } else {
      pooling_mode = exclusive ? PoolingMode::kAverageExclusive
                               : (PoolingMode::kAverageInclusive);
    }
    std::vector<int> k0x0 = {0, 0};
    std::vector<int> k1x1 = {1, 1};
    std::vector<int> k1x1_2 = {1, 1};
    std::vector<int> k3x3 = {3, 3};
    ScopedPoolingDescriptor pool_desc;
    ScopedActivationDescriptor act_desc;
    ScopedTensorDescriptor out_pool_desc;
    ScopedTensorDescriptor input_desc;
    hipdnnPoolingDescriptor_t cudnn_pool_desc =
        pool_desc.descriptor(pooling_mode, k3x3, k1x1, k1x1);

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize<int>(input->dims()));
    hipdnnTensorDescriptor_t pool_out_desc = out_pool_desc.descriptor<T>(
        layout, framework::vectorize<int>(input->dims()));

    hipdnnDataType_t cudnn_dtype = CudnnDataType<T>::type;
    hipdnnTensorDescriptor_t* out_desc = new hipdnnTensorDescriptor_t[4];
    hipdnnFilterDescriptor_t* filter_desc = new hipdnnFilterDescriptor_t[4];
    hipdnnTensorDescriptor_t* bias_desc = new hipdnnTensorDescriptor_t[4];
    hipdnnTensorDescriptor_t* in_desc = new hipdnnTensorDescriptor_t[4];
    hipdnnConvolutionDescriptor_t* conv_desc =
        new hipdnnConvolutionDescriptor_t[4];
    for (int i = 0; i < 4; ++i) {
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateFilterDescriptor(&filter_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&bias_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&in_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateTensorDescriptor(&out_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnCreateConvolutionDescriptor(&conv_desc[i]));
    }

    std::vector<std::vector<int>> filter_dims;
    std::vector<std::vector<int>> bias_dims;
    std::vector<std::vector<int>> in_dims;
    std::vector<std::vector<int>> out_dims;
    std::vector<std::vector<int>> in_strides;
    std::vector<std::vector<int>> out_strides;
    std::vector<std::vector<int>> bias_strides;

    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
    int n = in_dim[0];
    int h = in_dim[2];
    int w = in_dim[3];
    int oc = output->dims()[1];

    hipdnnDataType_t compute_type = (cudnn_dtype == HIPDNN_DATA_DOUBLE)
                                       ? HIPDNN_DATA_DOUBLE
                                       : HIPDNN_DATA_FLOAT;

    for (int i = 0; i < 4; ++i) {
      filter_dims.push_back(framework::vectorize<int>(filters[i]->dims()));
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetFilterNdDescriptor(
          filter_desc[i], cudnn_dtype, format, 4, filter_dims[i].data()));
      bias_dims.push_back({1, filter_dims[i][0], 1, 1});
      bias_strides.push_back({filter_dims[i][0], 1, 1, 1});
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
          bias_desc[i], cudnn_dtype, 4, bias_dims[i].data(),
          bias_strides[i].data()));
      in_dims.push_back({n, filter_dims[i][1], h, w});
      out_dims.push_back({n, filter_dims[i][0], h, w});
      in_strides.push_back({filter_dims[i][1] * h * w, h * w, w, 1});
      out_strides.push_back({oc * h * w, h * w, w, 1});

      if (i < 2) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::hipdnnSetConvolutionNdDescriptor(
                conv_desc[i], 2, k0x0.data(), k1x1.data(), k1x1.data(),
                HIPDNN_CROSS_CORRELATION, compute_type));
      } else {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::hipdnnSetConvolutionNdDescriptor(
                conv_desc[i], 2, k1x1.data(), k1x1.data(), k1x1.data(),
                HIPDNN_CROSS_CORRELATION, compute_type));
      }
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnSetConvolutionMathType(conv_desc[i],
                                                         HIPDNN_DEFAULT_MATH));
#if CUDNN_VERSION >= 11000
      if (!platform::allow_tf32_cudnn) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::hipdnnSetConvolutionMathType(conv_desc[i],
                                                           CUDNN_FMA_MATH));
      }
#endif  // CUDA_VERSION >= 11000
    }
    in_dims[2][1] *= 2;
    in_strides[2][0] = oc * h * w;
    out_strides[2][0] = filter_dims[2][0] * h * w;  // this out is continuous.
    in_strides[3][0] = filter_dims[2][0] * h * w;
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnSetConvolutionGroupCount(conv_desc[2], 2));

    hipdnnConvolutionFwdAlgo_t algo[4];
    auto handle = dev_ctx.cudnn_handle();
    size_t workspace_size_in_bytes = 0;  // final workspace to allocate.

    size_t workspace_size_limit = 0;
    if (FLAGS_conv_workspace_size_limit > 0 || user_workspace_size > 0) {
      int64_t max_user_size =
          std::min(static_cast<int64_t>(FLAGS_conv_workspace_size_limit),
                   user_workspace_size);
      workspace_size_limit = max_user_size * 1024 * 1024;
    }

    for (int i = 0; i < 4; ++i) {
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
          in_desc[i], cudnn_dtype, 4, in_dims[i].data(), in_strides[i].data()));
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
          out_desc[i], cudnn_dtype, 4, out_dims[i].data(),
          out_strides[i].data()));

      int perf_count;
      int best_algo_idx = 0;
      size_t tmp_size = 0;
      std::unique_ptr<hipdnnConvolutionFwdAlgoPerf_t[]> perf_results(
          new hipdnnConvolutionFwdAlgoPerf_t[kNUM_CUDNN_FWD_ALGS]);
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::cudnnGetConvolutionForwardAlgorithm_v7(
              handle, in_desc[i], filter_desc[i], conv_desc[i], out_desc[i],
              kNUM_CUDNN_FWD_ALGS, &perf_count, perf_results.get()));
      algo[i] = (perf_results.get())[best_algo_idx].algo;

      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnGetConvolutionForwardWorkspaceSize(
              handle, in_desc[i], filter_desc[i], conv_desc[i], out_desc[i],
              algo[i], &tmp_size));

      workspace_size_in_bytes = std::max(workspace_size_in_bytes, tmp_size);
    }
    hipdnnActivationDescriptor_t cudnn_act_desc =
        act_desc.descriptor<T>(activation);

    int oc0 = filter_dims[0][0];
    int oc1 = filter_dims[1][0] - filter_dims[2][1] * 2;
    int oc3 = filter_dims[3][0];
    int oc2 = oc - oc0 - oc1 - oc3;

    // branch1: pool + 1x1 conv
    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnPoolingForward(
        handle, cudnn_pool_desc, &alpha, cudnn_input_desc, input_data, &beta,
        pool_out_desc, temp_data));

    std::vector<const void*> in_datas;
    in_datas.push_back(static_cast<const void*>(temp_data));
    in_datas.push_back(static_cast<const void*>(input_data));
    in_datas.push_back(
        static_cast<const void*>(output_data + (oc0 + oc1) * h * w));
    T* temp2_data = temp_outs[1]->mutable_data<T>(
        framework::make_ddim(out_dims[2]), ctx.GetPlace());
    in_datas.push_back(static_cast<const void*>(temp2_data + oc2 * h * w));

    std::vector<void*> out_datas;
    out_datas.push_back(static_cast<void*>(output_data));
    out_datas.push_back(static_cast<void*>(output_data + oc0 * h * w));
    out_datas.push_back(static_cast<void*>(temp2_data));
    out_datas.push_back(
        static_cast<void*>(output_data + (oc0 + oc1 + oc2) * h * w));

    for (int i = 0; i < 4; ++i) {
      auto func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_CUDA_SUCCESS(
            platform::dynload::cudnnConvolutionBiasActivationForward(
                handle, &alpha, in_desc[i], in_datas[i], filter_desc[i],
                static_cast<const void*>(filters[i]->data<T>()), conv_desc[i],
                algo[i], cudnn_workspace, workspace_size_in_bytes, &beta,
                out_desc[i], out_datas[i], bias_desc[i],
                static_cast<const void*>(bias[i]->data<T>()), cudnn_act_desc,
                out_desc[i], out_datas[i]));
      };
      auto workspace_handle = dev_ctx.cudnn_workspace_handle();
      workspace_handle.RunFunc(func, workspace_size_in_bytes);
    }

    hipdnnTensorDescriptor_t x_desc;
    hipdnnTensorDescriptor_t y_desc;
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&x_desc));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnCreateTensorDescriptor(&y_desc));
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        x_desc, cudnn_dtype, 4, out_dims[3].data(), out_strides[2].data()));
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSetTensorNdDescriptor(
        y_desc, cudnn_dtype, 4, out_dims[3].data(), out_strides[3].data()));
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::cudnnTransformTensor(
        handle, CudnnDataType<T>::kOne(), x_desc,
        static_cast<const void*>(out_datas[2]), CudnnDataType<T>::kZero(),
        y_desc, static_cast<void*>(output_data + (oc0 + oc1) * h * w)));

    for (int i = 0; i < 4; ++i) {
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(in_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(out_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyFilterDescriptor(filter_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyTensorDescriptor(bias_desc[i]));
      PADDLE_ENFORCE_CUDA_SUCCESS(
          platform::dynload::hipdnnDestroyConvolutionDescriptor(conv_desc[i]));
    }
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(x_desc));
    PADDLE_ENFORCE_CUDA_SUCCESS(
        platform::dynload::hipdnnDestroyTensorDescriptor(y_desc));
  }
};
#endif

}  // namespace operators
}  // namespace paddle

#if CUDNN_VERSION >= 7100
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(conv2d_inception_fusion,
                        ops::CUDNNConvInceptionFusionOpKernel<float>,
                        ops::CUDNNConvInceptionFusionOpKernel<double>);
#endif
