#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/conv_cudnn_op_cache.h"
#include "paddle/fluid/platform/cudnn_helper.h"

DECLARE_uint64(conv_workspace_size_limit);

namespace paddle {
namespace operators {

#if CUDNN_VERSION >= 7100
using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using ScopedActivationDescriptor = platform::ScopedActivationDescriptor;
using DataLayout = platform::DataLayout;

using ScopedPoolingDescriptor = platform::ScopedPoolingDescriptor;
using PoolingMode = platform::PoolingMode;
template <typename T>
using ScalingParamType = typename platform::CudnnDataType<T>::ScalingParamType;

template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;

template <typename T>
class CUDNNConvInceptionFusionOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto* input = ctx.Input<Tensor>("Input");
    auto filters = ctx.MultiInput<framework::Tensor>("Filter");
    auto bias = ctx.MultiInput<framework::Tensor>("Bias");

    auto* output = ctx.Output<Tensor>("Output");
    auto temp_outs = ctx.MultiOutput<framework::Tensor>("TempOutput");

    const std::string pool_type = ctx.Attr<std::string>("pooling_type");
    const std::string activation = ctx.Attr<std::string>("activation");
    const bool exclusive = ctx.Attr<bool>("exclusive");

    int64_t user_workspace_size =
        static_cast<size_t>(ctx.Attr<int>("workspace_size_MB"));

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    T* temp_data = temp_outs[0]->mutable_data<T>(input->dims(), ctx.GetPlace());

    DataLayout layout = DataLayout::kNCHW;
    std::vector<int> in_dim = framework::vectorize2int(input->dims());

    // ------------------- cudnn descriptors ---------------------
    PoolingMode pooling_mode;
    if (pool_type == "max") {
      pooling_mode = PoolingMode::kMaximum;
    } else {
      pooling_mode = exclusive ? PoolingMode::kAverageExclusive
                               : (PoolingMode::kAverageInclusive);
    }
    std::vector<int> k0x0 = {0, 0};
    std::vector<int> k1x1 = {1, 1};
    std::vector<int> k1x1_2 = {1, 1};
    std::vector<int> k3x3 = {3, 3};
    ScopedPoolingDescriptor pool_desc;
    ScopedActivationDescriptor act_desc;
    ScopedTensorDescriptor out_pool_desc;
    ScopedTensorDescriptor input_desc;
    hipdnnPoolingDescriptor_t cudnn_pool_desc =
        pool_desc.descriptor(pooling_mode, k3x3, k1x1, k1x1);

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize2int(input->dims()));
    hipdnnTensorDescriptor_t pool_out_desc = out_pool_desc.descriptor<T>(
        layout, framework::vectorize2int(input->dims()));

    hipdnnDataType_t cudnn_dtype = CudnnDataType<T>::type;
    hipdnnTensorDescriptor_t* out_desc = new hipdnnTensorDescriptor_t[4];
    hipdnnFilterDescriptor_t* filter_desc = new hipdnnFilterDescriptor_t[4];
    hipdnnTensorDescriptor_t* bias_desc = new hipdnnTensorDescriptor_t[4];
    hipdnnTensorDescriptor_t* in_desc = new hipdnnTensorDescriptor_t[4];
    hipdnnConvolutionDescriptor_t* conv_desc =
        new hipdnnConvolutionDescriptor_t[4];
    for (int i = 0; i < 4; ++i) {
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateFilterDescriptor(&filter_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&bias_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&in_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateTensorDescriptor(&out_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnCreateConvolutionDescriptor(&conv_desc[i]));
    }

    std::vector<std::vector<int>> filter_dims;
    std::vector<std::vector<int>> bias_dims;
    std::vector<std::vector<int>> in_dims;
    std::vector<std::vector<int>> out_dims;
    std::vector<std::vector<int>> in_strides;
    std::vector<std::vector<int>> out_strides;
    std::vector<std::vector<int>> bias_strides;

    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
    int n = in_dim[0];
    int h = in_dim[2];
    int w = in_dim[3];
    int oc = output->dims()[1];

    hipdnnDataType_t compute_type = (cudnn_dtype == HIPDNN_DATA_DOUBLE)
                                       ? HIPDNN_DATA_DOUBLE
                                       : HIPDNN_DATA_FLOAT;

    for (int i = 0; i < 4; ++i) {
      filter_dims.push_back(framework::vectorize2int(filters[i]->dims()));
      CUDNN_ENFORCE(platform::dynload::hipdnnSetFilterNdDescriptor(
          filter_desc[i], cudnn_dtype, format, 4, filter_dims[i].data()));
      bias_dims.push_back({1, filter_dims[i][0], 1, 1});
      bias_strides.push_back({filter_dims[i][0], 1, 1, 1});
      CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
          bias_desc[i], cudnn_dtype, 4, bias_dims[i].data(),
          bias_strides[i].data()));
      in_dims.push_back({n, filter_dims[i][1], h, w});
      out_dims.push_back({n, filter_dims[i][0], h, w});
      in_strides.push_back({filter_dims[i][1] * h * w, h * w, w, 1});
      out_strides.push_back({oc * h * w, h * w, w, 1});

      if (i < 2) {
        CUDNN_ENFORCE(platform::dynload::hipdnnSetConvolutionNdDescriptor(
            conv_desc[i], 2, k0x0.data(), k1x1.data(), k1x1.data(),
            HIPDNN_CROSS_CORRELATION, compute_type));
      } else {
        CUDNN_ENFORCE(platform::dynload::hipdnnSetConvolutionNdDescriptor(
            conv_desc[i], 2, k1x1.data(), k1x1.data(), k1x1.data(),
            HIPDNN_CROSS_CORRELATION, compute_type));
      }
      CUDNN_ENFORCE(platform::dynload::hipdnnSetConvolutionMathType(
          conv_desc[i], HIPDNN_DEFAULT_MATH));
    }
    in_dims[2][1] *= 2;
    in_strides[2][0] = oc * h * w;
    out_strides[2][0] = filter_dims[2][0] * h * w;  // this out is continuous.
    in_strides[3][0] = filter_dims[2][0] * h * w;
    CUDNN_ENFORCE(
        platform::dynload::hipdnnSetConvolutionGroupCount(conv_desc[2], 2));

    hipdnnConvolutionFwdAlgo_t algo[4];
    auto handle = dev_ctx.cudnn_handle();
    size_t workspace_size_in_bytes = 0;  // final workspace to allocate.

    size_t workspace_size_limit = 0;
    if (FLAGS_conv_workspace_size_limit > 0 || user_workspace_size > 0) {
      int64_t max_user_size =
          std::min(static_cast<int64_t>(FLAGS_conv_workspace_size_limit),
                   user_workspace_size);
      workspace_size_limit = max_user_size * 1024 * 1024;
    }

    for (int i = 0; i < 4; ++i) {
      CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
          in_desc[i], cudnn_dtype, 4, in_dims[i].data(), in_strides[i].data()));
      CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
          out_desc[i], cudnn_dtype, 4, out_dims[i].data(),
          out_strides[i].data()));
      CUDNN_ENFORCE(platform::dynload::hipdnnGetConvolutionForwardAlgorithm(
          handle, in_desc[i], filter_desc[i], conv_desc[i], out_desc[i],
          HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, workspace_size_limit,
          &algo[i]));
      size_t tmp_size = 0;
      CUDNN_ENFORCE(platform::dynload::hipdnnGetConvolutionForwardWorkspaceSize(
          handle, in_desc[i], filter_desc[i], conv_desc[i], out_desc[i],
          algo[i], &tmp_size));
      workspace_size_in_bytes = std::max(workspace_size_in_bytes, tmp_size);
    }
    hipdnnActivationDescriptor_t cudnn_act_desc =
        act_desc.descriptor<T>(activation);

    int oc0 = filter_dims[0][0];
    int oc1 = filter_dims[1][0] - filter_dims[2][1] * 2;
    int oc3 = filter_dims[3][0];
    int oc2 = oc - oc0 - oc1 - oc3;

    // branch1: pool + 1x1 conv
    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    CUDNN_ENFORCE(platform::dynload::hipdnnPoolingForward(
        handle, cudnn_pool_desc, &alpha, cudnn_input_desc, input_data, &beta,
        pool_out_desc, temp_data));

    std::vector<const void*> in_datas;
    in_datas.push_back(static_cast<const void*>(temp_data));
    in_datas.push_back(static_cast<const void*>(input_data));
    in_datas.push_back(
        static_cast<const void*>(output_data + (oc0 + oc1) * h * w));
    T* temp2_data = temp_outs[1]->mutable_data<T>(
        framework::make_ddim(out_dims[2]), ctx.GetPlace());
    in_datas.push_back(static_cast<const void*>(temp2_data + oc2 * h * w));

    std::vector<void*> out_datas;
    out_datas.push_back(static_cast<void*>(output_data));
    out_datas.push_back(static_cast<void*>(output_data + oc0 * h * w));
    out_datas.push_back(static_cast<void*>(temp2_data));
    out_datas.push_back(
        static_cast<void*>(output_data + (oc0 + oc1 + oc2) * h * w));

    for (int i = 0; i < 4; ++i) {
      auto func = [&](void* cudnn_workspace) {
        CUDNN_ENFORCE(platform::dynload::cudnnConvolutionBiasActivationForward(
            handle, &alpha, in_desc[i], in_datas[i], filter_desc[i],
            static_cast<const void*>(filters[i]->data<T>()), conv_desc[i],
            algo[i], cudnn_workspace, workspace_size_in_bytes, &beta,
            out_desc[i], out_datas[i], bias_desc[i],
            static_cast<const void*>(bias[i]->data<T>()), cudnn_act_desc,
            out_desc[i], out_datas[i]));
      };
      auto workspace_handle = dev_ctx.cudnn_workspace_handle();
      workspace_handle.RunFunc(func, workspace_size_in_bytes);
    }

    hipdnnTensorDescriptor_t x_desc;
    hipdnnTensorDescriptor_t y_desc;
    CUDNN_ENFORCE(platform::dynload::hipdnnCreateTensorDescriptor(&x_desc));
    CUDNN_ENFORCE(platform::dynload::hipdnnCreateTensorDescriptor(&y_desc));
    CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
        x_desc, cudnn_dtype, 4, out_dims[3].data(), out_strides[2].data()));
    CUDNN_ENFORCE(platform::dynload::hipdnnSetTensorNdDescriptor(
        y_desc, cudnn_dtype, 4, out_dims[3].data(), out_strides[3].data()));
    CUDNN_ENFORCE(platform::dynload::cudnnTransformTensor(
        handle, CudnnDataType<T>::kOne(), x_desc,
        static_cast<const void*>(out_datas[2]), CudnnDataType<T>::kZero(),
        y_desc, static_cast<void*>(output_data + (oc0 + oc1) * h * w)));

    for (int i = 0; i < 4; ++i) {
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(in_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(out_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyFilterDescriptor(filter_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyTensorDescriptor(bias_desc[i]));
      CUDNN_ENFORCE(
          platform::dynload::hipdnnDestroyConvolutionDescriptor(conv_desc[i]));
    }
    CUDNN_ENFORCE(platform::dynload::hipdnnDestroyTensorDescriptor(x_desc));
    CUDNN_ENFORCE(platform::dynload::hipdnnDestroyTensorDescriptor(y_desc));
  }
};
#endif

}  // namespace operators
}  // namespace paddle

#if CUDNN_VERSION >= 7100
namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(conv2d_inception_fusion,
                        ops::CUDNNConvInceptionFusionOpKernel<float>,
                        ops::CUDNNConvInceptionFusionOpKernel<double>);
#endif
