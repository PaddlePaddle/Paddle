#include "hip/hip_runtime.h"
//   Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <string>

#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/operators/fused/fused_seqpool_cvm_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"

namespace paddle {
namespace operators {

template <typename T>
using Vector = framework::Vector<T>;

#define CUDA_KERNEL_LOOP(i, n)                                  \
  for (auto i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

// normal
template <typename T>
__global__ void FusedSeqpoolKernelNormal(const size_t N,
                                         T **input_values,
                                         T **seqpool_output_values,
                                         size_t **lods_values,
                                         const int batch_size,
                                         const int embedding_size,
                                         const float pad_value) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    auto &start = *(lods_values[x] + y);
    auto &end = *(lods_values[x] + y + 1);

    T val = static_cast<T>(pad_value);
    for (auto k = start; k < end; ++k) {
      val += *(input_values[x] + k * embedding_size + offset);
    }
    *(seqpool_output_values[x] + y * embedding_size + offset) = val;
  }
}

// join need show click input
template <typename T>
__global__ void FusedCVMKernelWithCVM(const size_t N,
                                      T **output_values,
                                      T **seqpool_output_values,
                                      const int batch_size,
                                      const int embedding_size,
                                      const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    if (offset == 0) {         // show
      *(output_values[x] + y * embedding_size) =
          log(*(seqpool_output_values[x] + y * embedding_size) + 1);
    } else if (offset == 1) {  // click
      *(output_values[x] + y * embedding_size + offset) =
          log(*(seqpool_output_values[x] + y * embedding_size + 1) + 1) -
          log(*(seqpool_output_values[x] + y * embedding_size) + 1);
    } else {
      *(output_values[x] + y * embedding_size + offset) =
          *(seqpool_output_values[x] + y * embedding_size + offset);
    }
  }
}

// update not need show click input
template <typename T>
__global__ void FusedCVMKernelNoCVM(const size_t N,
                                    T **output_values,
                                    T **seqpool_output_values,
                                    const int batch_size,
                                    const int no_cvm_embedding_size,
                                    const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / no_cvm_embedding_size;
    int offset = i % no_cvm_embedding_size;
    int x = key / batch_size;  // slot id
    int y = key % batch_size;  // ins id
    // no cvm
    *(output_values[x] + y * no_cvm_embedding_size + offset) =
        *(seqpool_output_values[x] + y * (no_cvm_embedding_size + cvm_offset) +
          offset + cvm_offset);
  }
}

template <typename T>
void FusedSeqpoolCVM(const framework::ExecutionContext
                         &ctx,  // const paddle::platform::Place &place,
                     const std::vector<const T *> &input_data,
                     const std::vector<T *> &output_data,
                     const std::vector<T *> &seqpool_output_data,
                     std::vector<const size_t *> lods,
                     const int batch_size,
                     const int slot_num,
                     const int embedding_size,
                     const float padding_value,
                     const bool use_cvm,
                     const int cvm_offset) {
  auto stream = ctx.template device_context<phi::GPUContext>().stream();
  auto &dev_ctx = ctx.template device_context<phi::GPUContext>();
  size_t total_ptr_len = input_data.size() + output_data.size() +
                         seqpool_output_data.size() + lods.size();
  auto temp_ptr =
      memory::AllocShared(ctx.GetPlace(), total_ptr_len * sizeof(void *));
  void *ptr = temp_ptr->ptr();

#ifdef PADDLE_WITH_HIP
  T **gpu_input_values = reinterpret_cast<T **>(temp_ptr->ptr());
  platform::GpuMemcpyAsync(gpu_input_values,
                           input_data.data(),
                           input_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);
  T **gpu_output_values =
      reinterpret_cast<T **>(&gpu_input_values[input_data.size()]);
  platform::GpuMemcpyAsync(gpu_output_values,
                           output_data.data(),
                           output_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);
  T **gpu_seqpool_output_values =
      reinterpret_cast<T **>(&gpu_output_values[output_data.size()]);
  platform::GpuMemcpyAsync(gpu_seqpool_output_values,
                           seqpool_output_data.data(),
                           seqpool_output_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);
  size_t **lods_values = reinterpret_cast<size_t **>(
      &gpu_seqpool_output_values[seqpool_output_data.size()]);
  platform::GpuMemcpyAsync(lods_values,
                           lods.data(),
                           lods.size() * sizeof(size_t *),
                           hipMemcpyHostToDevice,
                           stream);
#else
  T **gpu_input_values = reinterpret_cast<T **>(temp_ptr->ptr());
  platform::GpuMemcpyAsync(gpu_input_values,
                           input_data.data(),
                           input_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);
  T **gpu_output_values =
      reinterpret_cast<T **>(&gpu_input_values[input_data.size()]);
  platform::GpuMemcpyAsync(gpu_output_values,
                           output_data.data(),
                           output_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);
  T **gpu_seqpool_output_values =
      reinterpret_cast<T **>(&gpu_output_values[output_data.size()]);
  platform::GpuMemcpyAsync(gpu_seqpool_output_values,
                           seqpool_output_data.data(),
                           seqpool_output_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);
  size_t **lods_values = reinterpret_cast<size_t **>(
      &gpu_seqpool_output_values[seqpool_output_data.size()]);
  platform::GpuMemcpyAsync(lods_values,
                           lods.data(),
                           lods.size() * sizeof(size_t *),
                           hipMemcpyHostToDevice,
                           stream);
#endif

  size_t N = static_cast<size_t>(batch_size * slot_num * embedding_size);
  platform::GpuLaunchConfig config = platform::GetGpuLaunchConfig1D(dev_ctx, N);
  // first sum pool
  FusedSeqpoolKernelNormal<<<config.block_per_grid.x,
                             config.thread_per_block.x,
                             0,
                             stream>>>(N,
                                       gpu_input_values,
                                       gpu_seqpool_output_values,
                                       lods_values,
                                       batch_size,
                                       embedding_size,
                                       padding_value);
  // second log
  if (use_cvm) {
    FusedCVMKernelWithCVM<<<config.block_per_grid.x,
                            config.thread_per_block.x,
                            0,
                            stream>>>(N,
                                      gpu_output_values,
                                      gpu_seqpool_output_values,
                                      batch_size,
                                      embedding_size,
                                      cvm_offset);
  } else {
    // not need show click input
    N = static_cast<size_t>(batch_size * slot_num *
                            (embedding_size - cvm_offset));
    platform::GpuLaunchConfig config =
        platform::GetGpuLaunchConfig1D(dev_ctx, N);
    FusedCVMKernelNoCVM<<<config.block_per_grid.x,
                          config.thread_per_block.x,
                          0,
                          stream>>>(N,
                                    gpu_output_values,
                                    gpu_seqpool_output_values,
                                    batch_size,
                                    (embedding_size - cvm_offset),
                                    cvm_offset);
  }
}

// join grad
template <typename T>
__global__ void FusedSeqpoolCVMGradKernelWithCVM(const size_t N,
                                                 T **out_grads_values,
                                                 T **in_grads_values,
                                                 T **cvm_values,
                                                 size_t **lods_values,
                                                 const int batch_size,
                                                 const int embedding_size,
                                                 const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;  // embedx offset
    int x = key / batch_size;         // slot id
    int y = key % batch_size;         // ins id

    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] + y * embedding_size + offset);

    auto &start = *(lods_values[x] + y);
    auto &end = *(lods_values[x] + y + 1);
    for (auto k = start; k < end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// join only show not has click
template <typename T>
__global__ void FusedSeqpoolCVMGradKernelWithShow(const size_t N,
                                                  T **out_grads_values,
                                                  T **in_grads_values,
                                                  T **cvm_values,
                                                  size_t **lods_values,
                                                  const int batch_size,
                                                  const int embedding_size,
                                                  const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;  // embedx offset
    int x = key / batch_size;         // slot id
    int y = key % batch_size;         // ins id

    T &val =
        (offset < cvm_offset)
            ? *(cvm_values[x] + y * cvm_offset + offset)
            : *(out_grads_values[x] + y * (embedding_size - 1) + offset - 1);

    auto &start = *(lods_values[x] + y);
    auto &end = *(lods_values[x] + y + 1);
    for (auto k = start; k < end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

// update grad
template <typename T>
__global__ void FusedSeqpoolCVMGradKernelNoCVM(const size_t N,
                                               T **out_grads_values,
                                               T **in_grads_values,
                                               T **cvm_values,
                                               size_t **lods_values,
                                               const int batch_size,
                                               const int embedding_size,
                                               const int cvm_offset) {
  CUDA_KERNEL_LOOP(i, N) {
    int key = i / embedding_size;
    int offset = i % embedding_size;  // embedx offset
    int x = key / batch_size;         // slot id
    int y = key % batch_size;         // ins id

    T &val = (offset < cvm_offset)
                 ? *(cvm_values[x] + y * cvm_offset + offset)
                 : *(out_grads_values[x] + y * (embedding_size - cvm_offset) +
                     offset - cvm_offset);

    auto &start = *(lods_values[x] + y);
    auto &end = *(lods_values[x] + y + 1);
    for (auto k = start; k < end; ++k) {
      *(in_grads_values[x] + k * embedding_size + offset) = val;
    }
  }
}

template <typename T>
void FusedSeqpoolCVMGrad(const framework::ExecutionContext &ctx,
                         const std::vector<const T *> &out_grads_data,
                         const std::vector<T *> &in_grads_data,
                         const std::vector<const T *> &cvm_data,
                         const std::vector<const size_t *> &lods,
                         const int batch_size,
                         const int slot_num,
                         const int embedding_size,
                         const bool use_cvm,
                         const int cvm_offset) {
  auto stream = ctx.template device_context<phi::GPUContext>().stream();
  auto &dev_ctx = ctx.template device_context<phi::GPUContext>();
  size_t total_ptr_len = out_grads_data.size() + in_grads_data.size() +
                         cvm_data.size() + lods.size();
  auto temp_ptr =
      memory::AllocShared(ctx.GetPlace(), total_ptr_len * sizeof(void *));
#ifdef PADDLE_WITH_HIP
  T **gpu_out_grads_values = reinterpret_cast<T **>(temp_ptr->ptr());
  platform::GpuMemcpyAsync(gpu_out_grads_values,
                           out_grads_data.data(),
                           out_grads_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);

  T **gpu_in_grads_values =
      reinterpret_cast<T **>(&gpu_out_grads_values[out_grads_data.size()]);
  platform::GpuMemcpyAsync(gpu_in_grads_values,
                           in_grads_data.data(),
                           in_grads_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);

  T **gpu_cvm_values =
      reinterpret_cast<T **>(&gpu_in_grads_values[in_grads_data.size()]);
  platform::GpuMemcpyAsync(gpu_cvm_values,
                           cvm_data.data(),
                           cvm_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);

  size_t **lods_values =
      reinterpret_cast<size_t **>(&gpu_cvm_values[cvm_data.size()]);
  platform::GpuMemcpyAsync(lods_values,
                           lods.data(),
                           lods.size() * sizeof(size_t *),
                           hipMemcpyHostToDevice,
                           stream);
#else
  T **gpu_out_grads_values = reinterpret_cast<T **>(temp_ptr->ptr());
  platform::GpuMemcpyAsync(gpu_out_grads_values,
                           out_grads_data.data(),
                           out_grads_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);

  T **gpu_in_grads_values =
      reinterpret_cast<T **>(&gpu_out_grads_values[out_grads_data.size()]);
  platform::GpuMemcpyAsync(gpu_in_grads_values,
                           in_grads_data.data(),
                           in_grads_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);

  T **gpu_cvm_values =
      reinterpret_cast<T **>(&gpu_in_grads_values[in_grads_data.size()]);
  platform::GpuMemcpyAsync(gpu_cvm_values,
                           cvm_data.data(),
                           cvm_data.size() * sizeof(T *),
                           hipMemcpyHostToDevice,
                           stream);

  size_t **lods_values =
      reinterpret_cast<size_t **>(&gpu_cvm_values[cvm_data.size()]);
  platform::GpuMemcpyAsync(lods_values,
                           lods.data(),
                           lods.size() * sizeof(size_t *),
                           hipMemcpyHostToDevice,
                           stream);
#endif

  size_t N = static_cast<size_t>(batch_size * slot_num * embedding_size);
  auto config = platform::GetGpuLaunchConfig1D(dev_ctx, N);
  if (use_cvm) {
    // join grad
    FusedSeqpoolCVMGradKernelWithCVM<<<config.block_per_grid.x,
                                       config.thread_per_block.x,
                                       0,
                                       stream>>>(N,
                                                 gpu_out_grads_values,
                                                 gpu_in_grads_values,
                                                 gpu_cvm_values,
                                                 lods_values,
                                                 batch_size,
                                                 embedding_size,
                                                 cvm_offset);
  } else {
    // update grad
    FusedSeqpoolCVMGradKernelNoCVM<<<config.block_per_grid.x,
                                     config.thread_per_block.x,
                                     0,
                                     stream>>>(N,
                                               gpu_out_grads_values,
                                               gpu_in_grads_values,
                                               gpu_cvm_values,
                                               lods_values,
                                               batch_size,
                                               embedding_size,
                                               cvm_offset);
  }
}

template <typename T>
class FusedSeqpoolCVMCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto inputs = ctx.MultiInput<LoDTensor>("X");
    auto outputs = ctx.MultiOutput<framework::Tensor>("Out");
    auto &dev_ctx = ctx.template device_context<phi::GPUContext>();
    const auto slot_size = inputs.size();
    std::vector<const float *> input_data(slot_size);
    std::vector<const size_t *> lods_data(slot_size);
    std::vector<T *> output_data(slot_size);

    std::vector<LoDTensor> seqpool_outputs(slot_size);
    std::vector<T *> seqpool_output_data(slot_size);

    auto padding_value = ctx.Attr<float>("pad_value");
    auto use_cvm = ctx.Attr<bool>("use_cvm");
    const int cvm_offset = ctx.Attr<int>("cvm_offset");

    int embedding_size = inputs[0]->numel() / inputs[0]->dims()[0];
    int batch_size = -1;
    std::vector<paddle::framework::MixVector<size_t> *> mix_lods_v(slot_size);

    for (size_t i = 0; i < slot_size; ++i) {
      const auto *input = inputs[i];

      Vector<size_t> lods;
      if (input->lod().size() != 0) {
        auto lod = input->lod();
        lods = lod[0];
      } else {
        lods.push_back(0);
        for (int i = 0; i < input->dims()[0]; i++) {
          lods.push_back(i + 1);
        }
      }
      int cur_batch_size =
          input->lod().size() ? input->lod()[0].size() - 1 : input->dims()[0];
      if (batch_size == -1) {
        batch_size = cur_batch_size;
      } else {
        PADDLE_ENFORCE_EQ(batch_size,
                          cur_batch_size,
                          platform::errors::PreconditionNotMet(
                              "The batch size of all input should be same, "
                              "please cheack, last batchsize is %d, current "
                              "batchsize is %d",
                              batch_size,
                              cur_batch_size));
      }
      input_data[i] = reinterpret_cast<const T *>(input->data<T>());

      auto *output = outputs[i];
      if (use_cvm) {
        output->Resize({batch_size, embedding_size});
      } else {
        output->Resize({batch_size, embedding_size - cvm_offset});
      }
      output_data[i] = reinterpret_cast<T *>(
          dev_ctx.Alloc<T>(output, output->numel() * sizeof(T)));
      mix_lods_v[i] = new paddle::framework::MixVector<size_t>(&lods);
      lods_data[i] = mix_lods_v[i]->CUDAData(ctx.GetPlace());
      seqpool_outputs[i].Resize({batch_size, embedding_size});
      seqpool_output_data[i] = reinterpret_cast<T *>(dev_ctx.Alloc<T>(
          &seqpool_outputs[i], seqpool_outputs[i].numel() * sizeof(T)));
    }

    FusedSeqpoolCVM(ctx,
                    input_data,
                    output_data,
                    seqpool_output_data,
                    lods_data,
                    batch_size,
                    slot_size,
                    embedding_size,
                    padding_value,
                    use_cvm,
                    cvm_offset);

    for (int i = 0; i < slot_size; i++) {
      delete mix_lods_v[i];
    }
  }
};

template <typename T>
class FusedSeqpoolCVMGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto out_grads = ctx.MultiInput<LoDTensor>(framework::GradVarName("Out"));
    auto in_grads = ctx.MultiOutput<LoDTensor>(framework::GradVarName("X"));
    auto *cvm = ctx.Input<LoDTensor>("CVM");
    auto &dev_ctx = ctx.template device_context<phi::GPUContext>();
    std::string pooltype = ctx.Attr<std::string>("pooltype");
    auto use_cvm = ctx.Attr<bool>("use_cvm");
    const int cvm_offset = ctx.Attr<int>("cvm_offset");

    const auto slot_size = in_grads.size();
    std::vector<const T *> out_grads_data(slot_size);
    std::vector<T *> in_grads_data(slot_size);
    std::vector<const T *> cvm_data(slot_size);
    std::vector<const size_t *> lods_data(slot_size);

    int embedding_size = in_grads[0]->numel() / in_grads[0]->dims()[0];
    int batch_size = -1;
    std::vector<paddle::framework::MixVector<size_t> *> mix_lods_v(slot_size);

    for (size_t i = 0; i < slot_size; ++i) {
      auto *in_grad = in_grads[i];

      Vector<size_t> lods;
      if (in_grad->lod().size() != 0) {
        auto lod = in_grad->lod();
        lods = lod[0];
      } else {
        lods.push_back(0);
        for (int i = 0; i < in_grad->dims()[0]; i++) {
          lods.push_back(i + 1);
        }
      }

      int cur_batch_size = in_grad->lod().size() ? in_grad->lod()[0].size() - 1
                                                 : in_grad->dims()[0];
      if (batch_size == -1) {
        batch_size = cur_batch_size;
      } else {
        PADDLE_ENFORCE_EQ(batch_size,
                          cur_batch_size,
                          platform::errors::PreconditionNotMet(
                              "The batch size of all input should be same, "
                              "please cheack, last batchsize is %d, current "
                              "batchsize is %d",
                              batch_size,
                              cur_batch_size));
      }

      auto *out_grad = out_grads[i];
      out_grads_data[i] = reinterpret_cast<const T *>(out_grad->data<T>());

      in_grads_data[i] = reinterpret_cast<T *>(
          dev_ctx.Alloc<T>(in_grad, in_grad->numel() * sizeof(T)));
      mix_lods_v[i] = new paddle::framework::MixVector<size_t>(&lods);
      lods_data[i] = mix_lods_v[i]->CUDAData(ctx.GetPlace());
      cvm_data[i] = reinterpret_cast<const T *>(cvm->data<T>());
    }
    FusedSeqpoolCVMGrad(ctx,
                        out_grads_data,
                        in_grads_data,
                        cvm_data,
                        lods_data,
                        batch_size,
                        slot_size,
                        embedding_size,
                        use_cvm,
                        cvm_offset);

    for (int i = 0; i < slot_size; i++) {
      delete mix_lods_v[i];
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(fused_seqpool_cvm,
                        ops::FusedSeqpoolCVMCUDAKernel<float>);

REGISTER_OP_CUDA_KERNEL(fused_seqpool_cvm_grad,
                        ops::FusedSeqpoolCVMGradCUDAKernel<float>);
