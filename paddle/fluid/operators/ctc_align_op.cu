#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <vector>
#include "paddle/fluid/operators/ctc_align_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void MergeAndDelCudaKernel(const int64_t num_token, const T* tokens,
                                      const size_t num_seq, size_t* lod0,
                                      const int blank, const int merge_repeated,
                                      size_t* out_lod0, T* output) {
  int ouput_idx = 0;
  out_lod0[0] = 0;

  for (int i = 0; i < num_seq; ++i) {
    T pre_token = -1;
    for (int j = lod0[i]; j < lod0[i + 1]; ++j) {
      if (tokens[j] != blank && !(merge_repeated && tokens[j] == pre_token)) {
        output[ouput_idx] = tokens[j];
        ++ouput_idx;
      }
      pre_token = tokens[j];
    }
    out_lod0[i + 1] = ouput_idx;
  }
}

template <typename T>
__global__ void PaddingMergeAndDelCudaKernel(
    const int64_t num_token, const T* tokens, const T* tokens_length,
    const int blank, const int merge_repeated, const int padding_value,
    const int64_t batch_size, T* output, T* output_length) {
  int ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind >= batch_size) return;
  int output_idx = ind * num_token;
  T prev_token = -1;
  for (int i = ind * num_token; i < ind * num_token + tokens_length[ind]; i++) {
    if ((unsigned)tokens[i] != blank &&
        !(merge_repeated && tokens[i] == prev_token)) {
      output[output_idx] = tokens[i];
      ++output_idx;
    }
    prev_token = tokens[i];
  }
  output_length[ind] = output_idx - ind * num_token;
  for (int i = output_idx; i < ind * num_token + num_token; i++) {
    output[i] = padding_value;
  }
}

template <typename T>
class CTCAlignOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto* input = ctx.Input<LoDTensor>("Input");
    auto* output = ctx.Output<LoDTensor>("Output");
    const int blank = ctx.Attr<int>("blank");
    const int merge_repeated =
        static_cast<int>(ctx.Attr<bool>("merge_repeated"));
    const T* tokens = input->data<T>();
    auto stream = ctx.cuda_device_context().stream();

    // tensor input which has no lod
    if (input->lod().empty()) {
      const int padding_value = ctx.Attr<int>("padding_value");
      auto input_dims = input->dims();
      T* output_data = output->mutable_data<T>({input_dims[0], input_dims[1]},
                                               ctx.GetPlace());
      auto* input_length = ctx.Input<LoDTensor>("InputLength");
      const T* input_length_data = input_length->data<T>();
      auto* output_length = ctx.Output<LoDTensor>("OutputLength");
      T* output_length_data =
          output_length->mutable_data<T>({input_dims[0], 1}, ctx.GetPlace());
      PaddingMergeAndDelCudaKernel<
          T><<<32, (input_dims[0] + 32 - 1) / 32, 0, stream>>>(
          input_dims[1], tokens, input_length_data, blank, merge_repeated,
          padding_value, input_dims[0], output_data, output_length_data);
    } else {
      const size_t level = 0;
      auto input_lod = framework::ToAbsOffset(input->lod());

      const int64_t num_tokens = input->dims()[0];
      const size_t num_seq = input_lod[level].size() - 1;

      // prepare a lod to record lod information while merging elements
      thrust::device_vector<size_t> dev_out_lod0(input_lod[level].size());
      size_t* dev_out_lod0_ptr = thrust::raw_pointer_cast(dev_out_lod0.data());

      // merge elements and delete blank
      T* output_data = output->mutable_data<T>({num_tokens, 1}, ctx.GetPlace());

      MergeAndDelCudaKernel<T><<<1, 1, 0, stream>>>(
          num_tokens, tokens, num_seq,
          input_lod[level].CUDAMutableData(ctx.GetPlace()), blank,
          merge_repeated, dev_out_lod0_ptr, output_data);

      // set output lod
      std::vector<size_t> host_out_lod0(dev_out_lod0.begin(),
                                        dev_out_lod0.end());
      framework::LoD out_lod;
      out_lod.push_back(host_out_lod0);
      output->set_lod(out_lod);

      // resize output dims
      output->Resize({static_cast<int64_t>(host_out_lod0.back()), 1});

      if (host_out_lod0.back() == 0) {
        output->Resize({1, 1});
        output->mutable_data<T>(ctx.GetPlace());
        math::SetConstant<platform::CUDADeviceContext, T> set_constant;
        set_constant(ctx.template device_context<platform::CUDADeviceContext>(),
                     output, -1);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(ctc_align, paddle::operators::CTCAlignOpCUDAKernel<int>,
                        paddle::operators::CTCAlignOpCUDAKernel<int64_t>);
