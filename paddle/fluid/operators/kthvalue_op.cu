#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/kthvalue_op.h"
#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/fluid/operators/top_k_v2_op.h"
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
#endif

namespace paddle {
namespace operators {

int getBlockSize(int col) {
  if (col > 512)
    return 1024;
  else if (col > 256 && col <= 512)
    return 512;
  else if (col > 128 && col <= 256)
    return 256;
  else if (col > 64 && col <= 128)
    return 128;
  else
    return 64;
}

template <typename T>
bool SortKthvalue(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor* input_tensor, const int64_t num_cols,
                  const int64_t num_rows, const int k,
                  framework::Tensor* out_tensor,
                  framework::Tensor* indices_tensor) {
  auto cu_stream = ctx.stream();
  framework::Tensor input_indices;
  const std::vector<int64_t> dims = {num_rows, num_cols};
  auto dim = phi::make_ddim(dims);
  input_indices.Resize(dim);
  input_indices.mutable_data<int64_t>(ctx.GetPlace());
  size_t temp_storage_bytes = -1;
  int block_size = getBlockSize(num_cols);
  unsigned int maxGridDimX = ctx.GetCUDAMaxGridDimSize()[0];
  unsigned int grid_size = num_rows < maxGridDimX
                               ? static_cast<unsigned int>(num_rows)
                               : maxGridDimX;
  InitIndex<int64_t><<<grid_size, block_size, 0, cu_stream>>>(
      input_indices.data<int64_t>(), num_rows, num_cols);
  hipcub::CountingInputIterator<int64_t> counting_iter(0);
  hipcub::TransformInputIterator<int64_t, SegmentOffsetIter,
                              hipcub::CountingInputIterator<int64_t>>
      segment_offsets_t(counting_iter, SegmentOffsetIter(num_cols));
  T* sorted_values_ptr;
  int64_t* sorted_indices_ptr;
  framework::Tensor temp_values, temp_indices;
  const T* input = input_tensor->data<T>();
  T* values = out_tensor->data<T>();
  int64_t* indices = indices_tensor->mutable_data<int64_t>(ctx.GetPlace());
  temp_values.Resize(dim);
  temp_indices.Resize(dim);
  sorted_values_ptr = temp_values.mutable_data<T>(ctx.GetPlace());
  sorted_indices_ptr = temp_indices.mutable_data<int64_t>(ctx.GetPlace());
  auto err = hipcub::DeviceSegmentedRadixSort::SortPairs(
      nullptr, temp_storage_bytes, input, sorted_values_ptr,
      input_indices.data<int64_t>(), sorted_indices_ptr, num_cols * num_rows,
      num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
      cu_stream);
#ifdef __HIPCC__
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, status: "
               << hipGetErrorString(err);
    return false;
  }
#else
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, status: "
               << hipGetErrorString(err);
    return false;
  }
#endif
  framework::Tensor temp_storage;
  temp_storage.mutable_data<uint8_t>(ctx.GetPlace(), temp_storage_bytes);

  err = hipcub::DeviceSegmentedRadixSort::SortPairs(
      temp_storage.data<uint8_t>(), temp_storage_bytes, input,
      sorted_values_ptr, input_indices.data<int64_t>(), sorted_indices_ptr,
      num_cols * num_rows, num_rows, segment_offsets_t, segment_offsets_t + 1,
      0, sizeof(T) * 8, cu_stream);
#ifdef __HIPCC__
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, "
               << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
#else
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, "
               << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
#endif
  auto& dev = *ctx.eigen_device();
  const Eigen::DSizes<Eigen::DenseIndex, 2> slice_indices{0, k - 1};
  const Eigen::DSizes<Eigen::DenseIndex, 2> slice_sizes{num_rows, 1};
  auto e_indices = framework::EigenMatrix<int64_t>::From(*indices_tensor, dim);
  auto e_tmp_indices = framework::EigenMatrix<int64_t>::From(
      static_cast<const framework::Tensor>(temp_indices));
  std::vector<int> odims = {static_cast<int>(num_rows), static_cast<int>(1)};
  dim = phi::make_ddim(odims);
  auto e_values = framework::EigenMatrix<T>::From(*out_tensor, dim);
  auto e_tmp_values = framework::EigenMatrix<T>::From(
      static_cast<const framework::Tensor>(temp_values));

  EigenSlice<std::decay_t<decltype(dev)>, int64_t, 2>::Eval(
      dev, e_indices, e_tmp_indices, slice_indices, slice_sizes);
  EigenSlice<std::decay_t<decltype(dev)>, T, 2>::Eval(
      dev, e_values, e_tmp_values, slice_indices, slice_sizes);
  return true;
}

template <typename DeviceContext, typename T>
class KthvalueOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* input = ctx.Input<framework::Tensor>("X");
    auto* output = ctx.Output<framework::Tensor>("Out");
    auto* indices = ctx.Output<framework::Tensor>("Indices");
    int k = static_cast<int>(ctx.Attr<int>("k"));
    int axis = static_cast<int>(ctx.Attr<int>("axis"));
    bool keepdim = static_cast<bool>(ctx.Attr<bool>("keepdim"));
    const auto& in_dims = input->dims();
    if (axis < 0) axis += in_dims.size();
    auto out_dims = output->dims();
    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    int64_t* indices_data = indices->mutable_data<int64_t>(ctx.GetPlace());

    if (axis == in_dims.size() - 1) {
      const int64_t& input_height =
          phi::product(phi::slice_ddim(in_dims, 0, in_dims.size() - 1));
      const int64_t& input_width = in_dims[in_dims.size() - 1];
      const auto& dev_ctx = ctx.cuda_device_context();
      PADDLE_ENFORCE_EQ(SortKthvalue<T>(dev_ctx, input, input_width,
                                        input_height, k, output, indices),
                        true, platform::errors::External(
                                  "KthvalueOP: Error when use cub sorting"));
      return;
    } else {
      std::vector<int> trans;
      for (int i = 0; i < axis; i++) {
        trans.emplace_back(i);
      }
      trans.emplace_back(in_dims.size() - 1);
      for (int i = axis + 1; i < in_dims.size() - 1; i++) {
        trans.emplace_back(i);
      }
      trans.emplace_back(axis);
      if (!keepdim) {
        std::vector<int> tmp_out_shape;
        for (int i = 0; i < axis; i++) {
          tmp_out_shape.emplace_back(in_dims[i]);
        }
        tmp_out_shape.emplace_back(1);
        for (int i = axis + 1; i < in_dims.size(); i++) {
          tmp_out_shape.emplace_back(in_dims[i]);
        }
        framework::DDim tmp_out_dims = phi::make_ddim(tmp_out_shape);
        output->Resize(tmp_out_dims);
        indices->Resize(tmp_out_dims);
      }
      framework::DDim trans_dims(in_dims);
      framework::DDim trans_out_dims(in_dims);
      for (int i = 0; i < trans.size(); i++) {
        trans_dims[i] = in_dims[trans[i]];
        trans_out_dims[i] = in_dims[trans[i]];
      }
      trans_out_dims[in_dims.size() - 1] = 1;
      framework::Tensor trans_input;
      trans_input.mutable_data<T>(trans_dims, ctx.GetPlace());
      int ndims = trans.size();
      const auto& dev_ctx = ctx.cuda_device_context();
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, *input,
                                                   &trans_input, trans);
      framework::Tensor trans_ind, trans_out;
      trans_ind.mutable_data<int64_t>(trans_out_dims, ctx.GetPlace());
      trans_out.mutable_data<T>(trans_out_dims, ctx.GetPlace());
      const int64_t input_height =
          phi::product(phi::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
      const int64_t input_width = trans_dims[trans_dims.size() - 1];
      PADDLE_ENFORCE_EQ(
          SortKthvalue<T>(dev_ctx, &trans_input, input_width, input_height, k,
                          &trans_out, &trans_ind),
          true,
          platform::errors::External("KthvalueOP: Error when use cub sorting"));
      TransCompute<platform::CUDADeviceContext, int64_t>(
          ndims, dev_ctx, trans_ind, indices, trans);
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, trans_out,
                                                   output, trans);
      if (!keepdim) {
        output->Resize(out_dims);
        indices->Resize(out_dims);
      }
    }
  }
};

template <typename DeviceContext, typename T>
class KthvalueOpGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* x = context.Input<framework::Tensor>("X");
    auto* out_grad =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* indices = context.Input<framework::Tensor>("Indices");
    auto* x_grad =
        context.Output<framework::Tensor>(framework::GradVarName("X"));
    int axis = context.Attr<int>("axis");
    int k = static_cast<int>(context.Attr<int>("k"));
    const auto& in_dims = x->dims();
    auto out_dims = indices->dims();
    if (axis < 0) axis += in_dims.size();
    T* x_grad_data = x_grad->mutable_data<T>(context.GetPlace());
    const T* out_grad_data = out_grad->data<T>();
    const int64_t* indices_data = indices->data<int64_t>();
    int pre, n, post;
    GetDims(in_dims, axis, &pre, &n, &post);
    auto& dev_ctx = context.cuda_device_context();
    int block_size = getBlockSize(post * k);
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
    int grid_size = std::min(max_blocks, pre);
    AssignGradWithAxis<T><<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
        out_grad_data, indices_data, x_grad_data, pre, post, n, 1);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    kthvalue,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    kthvalue_grad,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                  int64_t>);
