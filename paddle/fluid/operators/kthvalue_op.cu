#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/kthvalue_op.h"
#include "paddle/fluid/operators/top_k_function_cuda.h"
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
#endif

namespace paddle {
namespace operators {

#define FIXED_BLOCK_DIM_BASE(dim, ...) \
  case (dim): {                        \
    constexpr auto kBlockDim = (dim);  \
    __VA_ARGS__;                       \
  } break

#define FIXED_BLOCK_DIM(...)                \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(32, ##__VA_ARGS__)

int ComputeBlockSize(int col) {
  if (col > 512)
    return 1024;
  else if (col > 256 && col <= 512)
    return 512;
  else if (col > 128 && col <= 256)
    return 256;
  else if (col > 64 && col <= 128)
    return 128;
  else
    return 64;
}

inline void GetDims(const framework::DDim& dim, int axis, int* pre, int* n,
                    int* post) {
  *pre = 1;
  *post = 1;
  *n = dim[axis];
  for (int i = 0; i < axis; ++i) {
    (*pre) *= dim[i];
  }
  for (int i = axis + 1; i < dim.size(); ++i) {
    (*post) *= dim[i];
  }
}

template <typename T>
bool SortKthvalue(const platform::CUDADeviceContext& ctx,
                  const framework::Tensor* input_tensor, const int64_t num_cols,
                  const int64_t num_rows, const int k,
                  framework::Tensor* out_tensor,
                  framework::Tensor* indices_tensor) {
  auto cu_stream = ctx.stream();

  framework::Tensor input_indices;
  const std::vector<int64_t> dims = {num_rows, num_cols};
  auto dim = framework::make_ddim(dims);
  input_indices.Resize(dim);
  input_indices.mutable_data<int64_t>(ctx.GetPlace());
  size_t temp_storage_bytes = -1;

  int block_size = ComputeBlockSize(num_cols);

  unsigned int maxGridDimX = ctx.GetCUDAMaxGridDimSize().x;
  // actually, int num_rows < max_grid_size
  unsigned int grid_size = num_rows < maxGridDimX
                               ? static_cast<unsigned int>(num_rows)
                               : maxGridDimX;
  // Init a index array
  InitIndex<int64_t><<<grid_size, block_size, 0, cu_stream>>>(
      input_indices.data<int64_t>(), num_rows, num_cols);

  // create iter for counting input
  hipcub::CountingInputIterator<int64_t> counting_iter(0);
  // segment_offset is used for move to next row
  hipcub::TransformInputIterator<int64_t, SegmentOffsetIter,
                              hipcub::CountingInputIterator<int64_t>>
      segment_offsets_t(counting_iter, SegmentOffsetIter(num_cols));

  T* sorted_values_ptr;
  int64_t* sorted_indices_ptr;

  framework::Tensor temp_values;
  framework::Tensor temp_indices;

  const T* input = input_tensor->data<T>();
  T* values = out_tensor->data<T>();
  int64_t* indices = indices_tensor->mutable_data<int64_t>(ctx.GetPlace());

  temp_values.Resize(dim);
  temp_indices.Resize(dim);
  sorted_values_ptr = temp_values.mutable_data<T>(ctx.GetPlace());
  sorted_indices_ptr = temp_indices.mutable_data<int64_t>(ctx.GetPlace());

  // Get temp storage buffer size, maybe can allocate a fixed buffer to save
  // time.
  auto err = hipcub::DeviceSegmentedRadixSort::SortPairs(
      nullptr, temp_storage_bytes, input, sorted_values_ptr,
      input_indices.data<int64_t>(), sorted_indices_ptr, num_cols * num_rows,
      num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
      cu_stream);
#ifdef __HIPCC__
  if (err != hipSuccess) {
    LOG(ERROR) << "TopKOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs to calculate "
                  "temp_storage_bytes, status: "
               << hipGetErrorString(err);
    return false;
  }
#else
  if (err != hipSuccess) {
    LOG(ERROR) << "TopKOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs to calculate "
                  "temp_storage_bytes, status: "
               << hipGetErrorString(err);
    return false;
  }
#endif
  framework::Tensor temp_storage;
  temp_storage.mutable_data<uint8_t>(ctx.GetPlace(), temp_storage_bytes);

  err = hipcub::DeviceSegmentedRadixSort::SortPairs(
      temp_storage.data<uint8_t>(), temp_storage_bytes, input,
      sorted_values_ptr, input_indices.data<int64_t>(), sorted_indices_ptr,
      num_cols * num_rows, num_rows, segment_offsets_t, segment_offsets_t + 1,
      0, sizeof(T) * 8, cu_stream);
#ifdef __HIPCC__
  if (err != hipSuccess) {
    LOG(ERROR) << "TopKOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs to "
                  "sort input, "
                  "temp_storage_bytes: "
               << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
#else
  if (err != hipSuccess) {
    LOG(ERROR) << "TopKOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs to "
                  "sort input, "
                  "temp_storage_bytes: "
               << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
#endif
  auto& dev = *ctx.eigen_device();
  // copy sliced data to output.
  const Eigen::DSizes<Eigen::DenseIndex, 2> slice_indices{0, k - 1};
  const Eigen::DSizes<Eigen::DenseIndex, 2> slice_sizes{num_rows, 1};
  auto e_indices = framework::EigenMatrix<int64_t>::From(*indices_tensor, dim);
  auto e_tmp_indices = framework::EigenMatrix<int64_t>::From(
      static_cast<const framework::Tensor>(temp_indices));

  std::vector<int> odims = {static_cast<int>(num_rows), static_cast<int>(1)};
  dim = framework::make_ddim(odims);
  auto e_values = framework::EigenMatrix<T>::From(*out_tensor, dim);
  auto e_tmp_values = framework::EigenMatrix<T>::From(
      static_cast<const framework::Tensor>(temp_values));

  EigenSlice<std::decay_t<decltype(dev)>, int64_t, 2>::Eval(
      dev, e_indices, e_tmp_indices, slice_indices, slice_sizes);
  EigenSlice<std::decay_t<decltype(dev)>, T, 2>::Eval(
      dev, e_values, e_tmp_values, slice_indices, slice_sizes);
  return true;
}

template <typename DeviceContext, typename T>
class KthvalueOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* input = ctx.Input<framework::Tensor>("X");
    auto* output = ctx.Output<framework::Tensor>("Out");
    auto* indices = ctx.Output<framework::Tensor>("Indices");

    // get the attributes
    int k = static_cast<int>(ctx.Attr<int>("k"));
    int axis = static_cast<int>(ctx.Attr<int>("axis"));
    bool keepdim = static_cast<bool>(ctx.Attr<bool>("keepdim"));

    // get the input dims
    const auto& in_dims = input->dims();
    // calcluate the real axis
    if (axis < 0) axis += in_dims.size();

    auto out_dims = output->dims();

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    int64_t* indices_data = indices->mutable_data<int64_t>(ctx.GetPlace());

    if (axis == in_dims.size() - 1) {
      const int64_t& input_height = framework::product(
          framework::slice_ddim(in_dims, 0, in_dims.size() - 1));
      const int64_t& input_width = in_dims[in_dims.size() - 1];
      const auto& dev_ctx = ctx.cuda_device_context();

      if (SortKthvalue<T>(dev_ctx, input, input_width, input_height, k, output,
                          indices)) {
        // Successed, return.
        return;
      } else {
        LOG(INFO) << "KthvalueOP: Some errors happened when use cub sorting";
      }
    } else {
      // first step, prepare the trans args for the tranpose
      std::vector<int> trans;
      for (int i = 0; i < axis; i++) {
        trans.emplace_back(i);
      }
      trans.emplace_back(in_dims.size() - 1);
      for (int i = axis + 1; i < in_dims.size() - 1; i++) {
        trans.emplace_back(i);
      }
      trans.emplace_back(axis);

      if (!keepdim) {
        std::vector<int> tmp_out_shape;
        for (int i = 0; i < axis; i++) {
          tmp_out_shape.emplace_back(in_dims[i]);
        }
        tmp_out_shape.emplace_back(1);
        for (int i = axis + 1; i < in_dims.size(); i++) {
          tmp_out_shape.emplace_back(in_dims[i]);
        }
        framework::DDim tmp_out_dims = framework::make_ddim(tmp_out_shape);
        output->Resize(tmp_out_dims);
        indices->Resize(tmp_out_dims);
      }

      framework::DDim trans_dims(in_dims);
      framework::DDim trans_out_dims(in_dims);
      for (int i = 0; i < trans.size(); i++) {
        trans_dims[i] = in_dims[trans[i]];
        trans_out_dims[i] = in_dims[trans[i]];
      }
      trans_out_dims[in_dims.size() - 1] = 1;

      // second step, tranpose the input
      framework::Tensor trans_input;
      trans_input.mutable_data<T>(trans_dims, ctx.GetPlace());
      int ndims = trans.size();
      const auto& dev_ctx = ctx.cuda_device_context();
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, *input,
                                                   &trans_input, trans);
      framework::Tensor trans_ind;
      trans_ind.mutable_data<int64_t>(trans_out_dims, ctx.GetPlace());
      framework::Tensor trans_out;
      trans_out.mutable_data<T>(trans_out_dims, ctx.GetPlace());

      const int64_t input_height = framework::product(
          framework::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
      const int64_t input_width = trans_dims[trans_dims.size() - 1];

      if (SortKthvalue<T>(dev_ctx, &trans_input, input_width, input_height, k,
                          &trans_out, &trans_ind)) {
        // last step, tranpose back the indices and output
        TransCompute<platform::CUDADeviceContext, int64_t>(
            ndims, dev_ctx, trans_ind, indices, trans);
        TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, trans_out,
                                                     output, trans);
        if (!keepdim) {
          output->Resize(out_dims);
          indices->Resize(out_dims);
        }
        return;
      } else {
        LOG(INFO) << "KthvalueOP: Some errors happened when use cub sorting";
      }

      // last step, tranpose back the indices and output
      TransCompute<platform::CUDADeviceContext, int64_t>(
          ndims, dev_ctx, trans_ind, indices, trans);
      TransCompute<platform::CUDADeviceContext, T>(ndims, dev_ctx, trans_out,
                                                   output, trans);
      if (!keepdim) {
        output->Resize(out_dims);
        indices->Resize(out_dims);
      }
    }
  }
};

#undef FIXED_BLOCK_DIM_BASE
#undef FIXED_BLOCK_DIM
template <typename DeviceContext, typename T>
class KthvalueOpGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument(
            "It must use CUDAPlace, you must check your device set."));
    auto* x = context.Input<framework::Tensor>("X");
    auto* out_grad =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto* indices = context.Input<framework::Tensor>("Indices");
    auto* x_grad =
        context.Output<framework::Tensor>(framework::GradVarName("X"));
    int axis = context.Attr<int>("axis");
    int k = static_cast<int>(context.Attr<int>("k"));

    const auto& in_dims = x->dims();
    auto out_dims = indices->dims();

    // get the real the axis and the k
    if (axis < 0) axis += in_dims.size();
    // allocate the cuda memory for the x_grad
    T* x_grad_data = x_grad->mutable_data<T>(context.GetPlace());
    const T* out_grad_data = out_grad->data<T>();
    const int64_t* indices_data = indices->data<int64_t>();

    int pre, n, post;
    GetDims(in_dims, axis, &pre, &n, &post);

    // calcluate the block and grid num
    auto& dev_ctx = context.cuda_device_context();
    int block_size = ComputeBlockSize(post * k);
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
    int grid_size = std::min(max_blocks, pre);

    // lanuch the cuda kernel to assign the grad
    AssignGradWithAxis<T><<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
        out_grad_data, indices_data, x_grad_data, pre, post, n, 1);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    kthvalue,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::KthvalueOpCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    kthvalue_grad,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::KthvalueOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                  int64_t>);
