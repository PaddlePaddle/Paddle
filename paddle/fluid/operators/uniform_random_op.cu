#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"
#include "paddle/fluid/operators/uniform_random_op.h"
namespace paddle {
namespace operators {

template <typename T>
struct UniformGenerator {
  T min_, max_;
  unsigned int seed_;
  T diag_val_;
  unsigned int diag_num_;
  unsigned int diag_step_;
  __host__ __device__ UniformGenerator(T min, T max, int seed, int diag_num,
                                       int diag_step, T diag_val)
      : min_(min),
        max_(max),
        seed_(seed),
        diag_num_(diag_num),
        diag_step_(diag_step),
        diag_val_(diag_val) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n);
    T out = dist(rng);
    unsigned int remainder = n % (diag_step_ + 1);
    if (remainder == 0 && diag_num_ > n / (diag_step_ + 1)) {
      out = diag_val_;
    }
    return out;
  }
};

template <typename T>
struct UniformGeneratorOffset {
  T min_, max_;
  unsigned int seed_;
  T diag_val_;
  unsigned int diag_num_;
  unsigned int diag_step_;
  int offset_;
  __host__ __device__ UniformGeneratorOffset(T min, T max, int seed,
                                             int diag_num, int diag_step,
                                             T diag_val, int offset)
      : min_(min),
        max_(max),
        seed_(seed),
        diag_num_(diag_num),
        diag_step_(diag_step),
        diag_val_(diag_val),
        offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n + offset_);
    T out = dist(rng);
    unsigned int remainder = n % (diag_step_ + 1);
    if (remainder == 0 && diag_num_ > n / (diag_step_ + 1)) {
      out = diag_val_;
    }
    return out;
  }
};

// It seems that Eigen::Tensor::random in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename T>
class GPUUniformRandomKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    framework::Tensor* tensor = nullptr;
    auto out_var = context.OutputVar("Out");
    std::vector<int64_t> new_shape;
    auto list_new_shape_tensor =
        context.MultiInput<framework::Tensor>("ShapeTensorList");
    if (list_new_shape_tensor.size() > 0 || context.HasInput("ShapeTensor")) {
      if (context.HasInput("ShapeTensor")) {
        auto* shape_tensor = context.Input<framework::Tensor>("ShapeTensor");
        new_shape = GetNewDataFromShapeTensor(shape_tensor);
      } else if (list_new_shape_tensor.size() > 0) {
        new_shape = GetNewDataFromShapeTensorList(list_new_shape_tensor);
      }
    }

    if (out_var->IsType<framework::SelectedRows>()) {
      auto* selected_rows = out_var->GetMutable<framework::SelectedRows>();
      tensor = selected_rows->mutable_value();
      auto shape = context.Attr<std::vector<int64_t>>("shape");
      if (!new_shape.empty()) shape = new_shape;
      tensor->Resize(framework::make_ddim(shape));
      selected_rows->mutable_rows()->reserve(shape[0]);
    } else if (out_var->IsType<framework::LoDTensor>()) {
      tensor = out_var->GetMutable<framework::LoDTensor>();
      if (!new_shape.empty()) tensor->Resize(framework::make_ddim(new_shape));
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Expected type of Output(out) in uniform_random_op must be Tensor, "
          "SelectedRows. But got "
          "unsupport type: %s.",
          framework::ToTypeName(out_var->Type())));
    }
    T* data = tensor->mutable_data<T>(context.GetPlace());
    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    bool seed_flag = false;
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
      seed_flag = true;
    }

    T min = static_cast<T>(context.Attr<float>("min"));
    T max = static_cast<T>(context.Attr<float>("max"));
    unsigned int diag_num =
        static_cast<unsigned int>(context.Attr<int>("diag_num"));
    unsigned int diag_step =
        static_cast<unsigned int>(context.Attr<int>("diag_step"));
    T diag_val = static_cast<T>(context.Attr<float>("diag_val"));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    int64_t size = tensor->numel();
    int device_id =
        BOOST_GET_CONST(platform::CUDAPlace, context.GetPlace()).GetDeviceId();
    auto gen_cuda = framework::GetDefaultCUDAGenerator(device_id);
    if (gen_cuda->GetIsInitPy() && seed_flag) {
      auto seed_offset = gen_cuda->IncrementOffset(1);
      int gen_offset = size * seed_offset.second;
      thrust::transform(
          index_sequence_begin, index_sequence_begin + size,
          thrust::device_ptr<T>(data),
          UniformGeneratorOffset<T>(min, max, seed_offset.first, diag_num,
                                    diag_step, diag_val, gen_offset));
    } else {
      thrust::transform(
          index_sequence_begin, index_sequence_begin + size,
          thrust::device_ptr<T>(data),
          UniformGenerator<T>(min, max, seed, diag_num, diag_step, diag_val));
    }
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(uniform_random,
                        paddle::operators::GPUUniformRandomKernel<float>,
                        paddle::operators::GPUUniformRandomKernel<double>);
REGISTER_OP_CUDA_KERNEL(uniform_random_batch_size_like,
                        paddle::operators::GPUUniformRandomKernel<float>,
                        paddle::operators::GPUUniformRandomKernel<double>);
