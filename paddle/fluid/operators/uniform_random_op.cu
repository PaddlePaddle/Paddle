#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/operator.h"

namespace paddle {
namespace operators {

template <typename T>
struct UniformGenerator {
  T min_, max_;
  unsigned int seed_;

  __host__ __device__ UniformGenerator(T min, T max, int seed)
      : min_(min), max_(max), seed_(seed) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n);
    return dist(rng);
  }
};

// It seems that Eigen::Tensor::random in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename T>
class GPUUniformRandomKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    framework::Tensor* tensor = nullptr;
    auto out_var = context.OutputVar("Out");
    if (out_var->IsType<framework::LoDTensor>()) {
      tensor = out_var->GetMutable<framework::LoDTensor>();
    } else if (out_var->IsType<framework::SelectedRows>()) {
      auto shape = context.Attr<std::vector<int64_t>>("shape");
      tensor = out_var->GetMutable<framework::SelectedRows>()->mutable_value();
      tensor->Resize(framework::make_ddim(shape));
    } else {
      PADDLE_THROW(
          "uniform_random_op's output only"
          "supports SelectedRows and LoDTensor");
    }
    T* data = tensor->mutable_data<T>(context.GetPlace());
    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
    }
    T min = static_cast<T>(context.Attr<float>("min"));
    T max = static_cast<T>(context.Attr<float>("max"));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    int64_t size = tensor->numel();
    thrust::transform(index_sequence_begin, index_sequence_begin + size,
                      thrust::device_ptr<T>(data),
                      UniformGenerator<T>(min, max, seed));
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(uniform_random,
                        paddle::operators::GPUUniformRandomKernel<float>,
                        paddle::operators::GPUUniformRandomKernel<double>);
REGISTER_OP_CUDA_KERNEL(uniform_random_batch_size_like,
                        paddle::operators::GPUUniformRandomKernel<float>,
                        paddle::operators::GPUUniformRandomKernel<double>);
