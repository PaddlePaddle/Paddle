#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <limits>
#include "paddle/fluid/operators/log_softmax_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/funcs/functors.h"

namespace paddle {
namespace operators {

#define LAUNCH_WARP_FORWAR_COMPUTE(near_greater_power_of_two)                \
  case near_greater_power_of_two:                                            \
    ComputeLogSoftmaxForwardInWarp<                                          \
        T, AccT, near_greater_power_of_two><<<blocks, threads, 0, stream>>>( \
        dst, src, outer_size, dim_size);                                     \
    break;

template <typename T, int KernelWarpSize>
__device__ __forceinline__ T WarpReduceSum(T value) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
    T sum_val = platform::CudaShuffleXorSync(0xFFFFFFFF, value, offset);
    value = value + sum_val;
  }
  return value;
}

template <typename T, int KernelWarpSize>
__device__ __forceinline__ T WarpReduceMax(T value) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
    T max_val = platform::CudaShuffleXorSync(0xFFFFFFFF, value, offset);
    value = max(value, max_val);
  }
  return value;
}

int GetNearGreaterPowerOfTwo(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) {
    ++log2_value;
  }
  return 1 << log2_value;
}

template <typename T, typename AccT, int NearGreaterPowerOfTwo>
__global__ void ComputeLogSoftmaxForwardInWarp(T *dst, const T *src,
                                               int batch_size,
                                               int element_count) {
  constexpr int near_greater_power_of_two = NearGreaterPowerOfTwo;
  constexpr int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  constexpr int warp_iter = near_greater_power_of_two / kernel_warp_size;
  int batch_id = blockDim.y * blockIdx.x + threadIdx.y;

  int thread_in_warp_idx = threadIdx.x;

  // 1.read data from global memory to registers
  AccT elements[warp_iter];
  // set effective_element_count as the num of elements when warps do effective
  // work
  // set effective_element_count as 0, when warps do ineffective work
  int effective_element_count = (batch_id < batch_size) ? element_count : 0;
  for (int it = 0; it < warp_iter; ++it) {
    int element_index = thread_in_warp_idx + it * kernel_warp_size;
    if (element_index < effective_element_count) {
      elements[it] =
          static_cast<AccT>(src[batch_id * element_count + element_index]);
    } else {
      elements[it] = -std::numeric_limits<AccT>::infinity();
    }
  }

  // 2.compute max_value. For each thread, loop all registers to find max
  AccT max_value = elements[0];
#pragma unroll
  for (int it = 1; it < warp_iter; ++it) {
    max_value = (max_value > elements[it]) ? max_value : elements[it];
  }
  max_value = WarpReduceMax<AccT, kernel_warp_size>(max_value);

  // 3.For each warp, accumulate all thread registers
  AccT sum = 0.0f;
#pragma unroll
  for (int it = 0; it < warp_iter; ++it) {
    sum += std::exp(elements[it] - max_value);
  }
  sum = WarpReduceSum<AccT, kernel_warp_size>(sum);

  // 4.store result.
  sum = std::log(sum);
#pragma unroll
  for (int it = 0; it < warp_iter; ++it) {
    int element_index = thread_in_warp_idx + it * kernel_warp_size;
    if (element_index < effective_element_count) {
      dst[batch_id * element_count + element_index] =
          static_cast<T>(elements[it] - max_value - sum);
    } else {
      break;
    }
  }
}

template <typename T, typename AccT>
void LaunchSoftmaxForwardForLastAxis(T *dst, const T *src, int dim_size,
                                     int outer_size, gpuStream_t stream) {
  int threads_per_block = 128;
  int near_greater_power_of_two = GetNearGreaterPowerOfTwo(dim_size);
  int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  int warps_per_block = (threads_per_block / kernel_warp_size);
  int blocks = (outer_size + warps_per_block - 1) / warps_per_block;
  dim3 threads(kernel_warp_size, warps_per_block, 1);

  switch (near_greater_power_of_two) {
    LAUNCH_WARP_FORWAR_COMPUTE(1);
    LAUNCH_WARP_FORWAR_COMPUTE(2);
    LAUNCH_WARP_FORWAR_COMPUTE(4);     // dim_size: 3~4
    LAUNCH_WARP_FORWAR_COMPUTE(8);     // dim_size: 5~8
    LAUNCH_WARP_FORWAR_COMPUTE(16);    // dim_size: 9~16
    LAUNCH_WARP_FORWAR_COMPUTE(32);    // dim_size: 17~32
    LAUNCH_WARP_FORWAR_COMPUTE(64);    // dim_size: 33~64
    LAUNCH_WARP_FORWAR_COMPUTE(128);   // dim_size 65~128
    LAUNCH_WARP_FORWAR_COMPUTE(256);   // dim_size 129~256
    LAUNCH_WARP_FORWAR_COMPUTE(512);   // dim_size 257~512
    LAUNCH_WARP_FORWAR_COMPUTE(1024);  // dim_size 513~1024

    default:
      break;
  }
}

// Returns the final item after reduce operation along block.x.
// Firstly, get shared memory(smem) offset, find the starting position for every
// y.
// Secondly, initialise every smem position with value 'val' of thread itself.
// Thirdly, apply standard reduction along x direction as below:
//
//   -> x direction
// [o o o o o o o o]    time 0
//  |     |/     /
//  |    /|    /
//  |  /  |  /
//  |/    |/
// [o o o o x x x x]    time 1
//  | |/ /
//  |/|/
// [o o x x x x x x]    time 2
//  |/
// [o x x x x x x x]    time 3
//
// Finally, return the first item.
// Imaging multiple reductions executed in paralell along y axis,
// Note that when blockDim.x is not 1, it's a EVEN number in all cases,
// and the size of shared memory is even as well.
template <typename T, template <typename> class Functor>
__forceinline__ __device__ T BlockReduceAlongDimX(T *shared, T val) {
  Functor<T> func;
  // This reduction is not Block-wise reduction, only reduce along block.x.
  // therefore the shared mem has offsets for different block.y.
  shared += threadIdx.y * blockDim.x;
  shared[threadIdx.x] = val;
  int offset = blockDim.x / 2;

  while (offset > 0) {
    __syncthreads();
    if (threadIdx.x < offset) {
      shared[threadIdx.x] =
          func(shared[threadIdx.x], shared[threadIdx.x + offset]);
    }
    offset /= 2;
  }
  __syncthreads();
  return shared[0];
}

template <typename T, typename AccT>
__global__ void LogSoftmaxForwardCUDAKernelNotLastAxis(
    T *output, const T *input, int outer_size, int dim_size, int inner_size) {
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<AccT *>(smem);

  const int outer_stride = inner_size * dim_size;
  const int dim_stride = inner_size;

  for (int x_id = blockIdx.x; x_id < outer_size; x_id += gridDim.x) {
    for (int y_id = blockIdx.y * blockDim.y + threadIdx.y; y_id < inner_size;
         y_id += blockDim.y * gridDim.y) {
      const int data_offset = x_id * outer_stride + y_id;
      // When blockDim.x==1, no block.x-reduction opetaions are needed.
      // And threadIdx.x is 0 all the time, so the for-loops below are literally
      // loops (No parallel executions). Loop all elements along axis and
      // calculate the Max, Sum and (input[id]-Max-log(Sum)) to get the final
      // log_softmax values along that axis.
      // 1. reduce max
      AccT max_value = -std::numeric_limits<AccT>::infinity();
      // For one thread, iterate all items it responsable for, and get
      // max_value.
      // If there are N threads, N max_value will be returned.
      for (int d = threadIdx.x; d < dim_size; d += blockDim.x) {
        const AccT value =
            static_cast<AccT>(input[data_offset + d * dim_stride]);
        max_value = phi::funcs::MaxFunctor<AccT>()(max_value, value);
      }
      // If there are more than 1 threads along block x, reduce all max_values
      // and get the global max_value, which is the max value along "axis".
      // If there is only one thread along block x, no need to reduce, as the
      // 'max_value' is the global max_value.
      if (blockDim.x > 1) {
        max_value = BlockReduceAlongDimX<AccT, phi::funcs::MaxFunctor>(
            sdata, max_value);
      }

      // 2. reduce sum
      AccT sum = 0;
      // Below is the same execution as '1. reduce max'
      for (int d = threadIdx.x; d < dim_size; d += blockDim.x) {
        sum += std::exp(static_cast<AccT>(input[data_offset + d * dim_stride]) -
                        max_value);
      }
      if (blockDim.x > 1) {
        sum = BlockReduceAlongDimX<AccT, phi::funcs::AddFunctor>(sdata, sum);
      }

      // 3. input-max-log_sum and write to output
      for (int d = threadIdx.x; d < dim_size; d += blockDim.x) {
        output[data_offset + d * dim_stride] = static_cast<T>(
            static_cast<AccT>(input[data_offset + d * dim_stride]) - max_value -
            std::log(sum));
      }
    }
  }
}

// block.y covers inner_size. Threads along the x axis process dim_size
// elements, and make sure not to exceed the 1024 threads per block.
// Note that dim_threads namely blockDim.x is either 1 or a even number.
inline dim3 GetBlockSize(int dim_size, int inner_size) {
  int inner_threads = inner_size;
  inner_threads = std::min(inner_threads, 1024);
  int dim_threads = 1;

  while (dim_threads * inner_threads <= 1024 && dim_threads <= dim_size) {
    dim_threads *= 2;
  }
  dim_threads /= 2;
  return dim3(dim_threads, inner_threads);
}

// First cover the y axis as many blocks as possible.
// Then cover the x axis as many blocks as possible,
// and make sure not to exceed the max_active_blocks.
inline dim3 GetGridSize(dim3 block, int max_active_blocks, int outer_size,
                        int dim_size, int inner_size) {
  int inner_blocks = (inner_size + block.y - 1) / block.y;
  if (inner_blocks > max_active_blocks) inner_blocks = max_active_blocks;

  int outer_blocks = (max_active_blocks + inner_blocks - 1) / inner_blocks;
  if (outer_blocks > outer_size) outer_blocks = outer_size;
  return dim3(outer_blocks, inner_blocks);
}

// When designing grid size and block size, priority is given to block size,
// and grid will be determined according to the maximum number of active blocks,
// which is set by as a experience value.
template <typename T, typename Kernel>
void ComputeLaunchConfigure(Kernel k, int outer_size, int dim_size,
                            int inner_size, dim3 &grid, dim3 &block,
                            int &shared_mem, int num_sm) {
  block = GetBlockSize(dim_size, inner_size);
  int block_threads = block.x * block.y;
  shared_mem = block.x == 1 ? 0 : block_threads * sizeof(T);
  int max_active_blocks = num_sm * 2;
  grid =
      GetGridSize(block, max_active_blocks, outer_size, dim_size, inner_size);
}

template <typename T, typename MPDType>
void LaunchLogSoftmaxForwardCUDAKernelNotLastAxis(T *output_data,
                                                  const T *input_data,
                                                  int outer_size, int dim_size,
                                                  int inner_size, int num_sm,
                                                  gpuStream_t stream) {
  int shared_mem;
  dim3 grid;
  dim3 block;

  ComputeLaunchConfigure<MPDType>(
      &LogSoftmaxForwardCUDAKernelNotLastAxis<T, MPDType>, outer_size, dim_size,
      inner_size, grid, block, shared_mem, num_sm);

  LogSoftmaxForwardCUDAKernelNotLastAxis<
      T, MPDType><<<grid, block, shared_mem, stream>>>(
      output_data, input_data, outer_size, dim_size, inner_size);
}

template <typename T>
class LogSoftmaxKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;

 public:
  void Compute(const framework::ExecutionContext &context) const override {
    const auto *x = context.Input<framework::Tensor>("X");
    auto *out = context.Output<framework::Tensor>("Out");
    const auto *input_data = x->data<T>();
    auto *output_data = out->mutable_data<T>(context.GetPlace());

    const int rank = x->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);

    int dim_size = x->dims()[axis];
    int inner_size = 1;
    for (int i = axis + 1; i < x->dims().size(); ++i) {
      inner_size *= x->dims()[i];
    }
    int outer_size = SizeToAxis(axis, x->dims());
    gpuStream_t stream = context.cuda_device_context().stream();
    int num_sm = context.cuda_device_context().GetSMCount();

    if (inner_size == 1 && dim_size <= 1024 && dim_size * sizeof(T) <= 4096) {
      LaunchSoftmaxForwardForLastAxis<T, MPDType>(output_data, input_data,
                                                  dim_size, outer_size, stream);
    } else {
      LaunchLogSoftmaxForwardCUDAKernelNotLastAxis<T, MPDType>(
          output_data, input_data, outer_size, dim_size, inner_size, num_sm,
          stream);
    }
  }
};

// Backward below
#define LAUNCH_WARP_BACKWARD_COMPUTE(near_greater_power_of_two)              \
  case near_greater_power_of_two:                                            \
    ComputeLogSoftmaxBackwardInWarp<                                         \
        T, AccT, near_greater_power_of_two><<<blocks, threads, 0, stream>>>( \
        output, grad_output, grad_input, outer_size, dim_size);              \
    break;

template <typename T, typename AccT, int NearGreaterPowerOfTwo>
__global__ void ComputeLogSoftmaxBackwardInWarp(const T *output,
                                                const T *grad_output,
                                                T *grad_input, int batch_size,
                                                int element_count) {
  constexpr int near_greater_power_of_two = NearGreaterPowerOfTwo;
  constexpr int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  constexpr int warp_iter = near_greater_power_of_two / kernel_warp_size;
  int batch_id = blockDim.y * blockIdx.x + threadIdx.y;

  int thread_in_warp_idx = threadIdx.x;

  // 1.read data from global memory to registers
  AccT output_register[warp_iter];
  AccT grad_output_register[warp_iter];
  int effective_element_count = (batch_id < batch_size) ? element_count : 0;
  for (int iter = 0; iter < warp_iter; ++iter) {
    int element_index = thread_in_warp_idx + iter * kernel_warp_size;
    if (element_index < effective_element_count) {
      output_register[iter] =
          static_cast<AccT>(output[batch_id * element_count + element_index]);
      grad_output_register[iter] = static_cast<AccT>(
          grad_output[batch_id * element_count + element_index]);
    } else {
      output_register[iter] = static_cast<AccT>(0);
      grad_output_register[iter] = static_cast<AccT>(0);
    }
  }

  // 2. For each warp, accumulate all thread registers
  AccT sum = grad_output_register[0];
#pragma unroll
  for (int iter = 1; iter < warp_iter; ++iter) {
    sum += grad_output_register[iter];
  }
  sum = WarpReduceSum<AccT, kernel_warp_size>(sum);

// 3. write result in grad_input
#pragma unroll
  for (int iter = 0; iter < warp_iter; ++iter) {
    int element_index = thread_in_warp_idx + iter * kernel_warp_size;
    if (element_index < effective_element_count) {
      grad_input[batch_id * element_count + element_index] = static_cast<T>(
          (grad_output_register[iter] - std::exp(output_register[iter]) * sum));
    }
  }
}

template <typename T, typename AccT>
void LaunchSoftmaxBackwardForLastAxis(T *grad_input, const T *grad_output,
                                      const T *output, int dim_size,
                                      int outer_size, gpuStream_t stream) {
  int threads_per_block = 128;
  int near_greater_power_of_two = GetNearGreaterPowerOfTwo(dim_size);
  int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  int warps_per_block = (threads_per_block / kernel_warp_size);
  int blocks = (outer_size + warps_per_block - 1) / warps_per_block;
  dim3 threads(kernel_warp_size, warps_per_block, 1);

  switch (near_greater_power_of_two) {
    LAUNCH_WARP_BACKWARD_COMPUTE(1);     // dim_size: 1
    LAUNCH_WARP_BACKWARD_COMPUTE(2);     // dim_size: 2
    LAUNCH_WARP_BACKWARD_COMPUTE(4);     // dim_size: 3~4
    LAUNCH_WARP_BACKWARD_COMPUTE(8);     // dim_size: 5~8
    LAUNCH_WARP_BACKWARD_COMPUTE(16);    // dim_size: 9~16
    LAUNCH_WARP_BACKWARD_COMPUTE(32);    // dim_size: 17~32
    LAUNCH_WARP_BACKWARD_COMPUTE(64);    // dim_size: 33~64
    LAUNCH_WARP_BACKWARD_COMPUTE(128);   // dim_size: 65~128
    LAUNCH_WARP_BACKWARD_COMPUTE(256);   // dim_size: 129~256
    LAUNCH_WARP_BACKWARD_COMPUTE(512);   // dim_size: 257~512
    LAUNCH_WARP_BACKWARD_COMPUTE(1024);  // dim_size: 513~1024

    default:
      break;
  }
}

template <typename T>
class LogSoftmaxGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;

 public:
  void Compute(const framework::ExecutionContext &context) const override {
    const auto *out = context.Input<framework::Tensor>("Out");
    const auto *d_out =
        context.Input<framework::Tensor>(framework::GradVarName("Out"));
    auto *d_x = context.Output<framework::Tensor>(framework::GradVarName("X"));

    const auto *out_data = out->data<T>();
    const auto *d_out_data = d_out->data<T>();
    auto *d_x_data = d_x->mutable_data<T>(context.GetPlace());

    const int rank = out->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);

    int dim_size = out->dims()[axis];
    int inner_size = 1;
    for (int i = axis + 1; i < out->dims().size(); ++i) {
      inner_size *= out->dims()[i];
    }
    int outer_size = SizeToAxis(axis, out->dims());
    gpuStream_t stream = context.cuda_device_context().stream();

    if (inner_size == 1 && dim_size <= 1024 && dim_size * sizeof(T) <= 4096) {
      LaunchSoftmaxBackwardForLastAxis<T, MPDType>(
          d_x_data, d_out_data, out_data, dim_size, outer_size, stream);
    } else {
      LogSoftmaxGradFunctor<platform::CUDADeviceContext, T>()(
          context.template device_context<platform::CUDADeviceContext>(), out,
          d_out, d_x, axis);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    log_softmax, ops::LogSoftmaxKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, plat::float16>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, plat::bfloat16>);
REGISTER_OP_CUDA_KERNEL(
    log_softmax_grad, ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, plat::float16>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, plat::bfloat16>);
