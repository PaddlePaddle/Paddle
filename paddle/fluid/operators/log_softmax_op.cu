#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <limits>
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/log_softmax_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"

namespace paddle {
namespace operators {

#define LAUNCH_WARP_FORWAR_COMPUTE(near_greater_power_of_two)                \
  case near_greater_power_of_two:                                            \
    ComputeLogSoftmaxForwardInWarp<                                          \
        T, AccT, near_greater_power_of_two><<<blocks, threads, 0, stream>>>( \
        dst, src, outer_size, dim_size);                                     \
    break;

template <typename T, int KernelWarpSize>
__device__ __forceinline__ T WarpReduceSum(T value) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
    T sum_val = platform::CudaShuffleXorSync(0xFFFFFFFF, value, offset);
    value = value + sum_val;
  }
  return value;
}

template <typename T, int KernelWarpSize>
__device__ __forceinline__ T WarpReduceMax(T value) {
#pragma unroll
  for (int offset = KernelWarpSize / 2; offset > 0; offset /= 2) {
    T max_val = platform::CudaShuffleXorSync(0xFFFFFFFF, value, offset);
    value = max(value, max_val);
  }
  return value;
}

int GetNearGreaterPowerOfTwo(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) {
    ++log2_value;
  }
  return 1 << log2_value;
}

template <typename T, typename AccT, int NearGreaterPowerOfTwo>
__global__ void ComputeLogSoftmaxForwardInWarp(T *dst, const T *src,
                                               int batch_size,
                                               int element_count) {
  constexpr int near_greater_power_of_two = NearGreaterPowerOfTwo;
  constexpr int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  constexpr int warp_iter = near_greater_power_of_two / kernel_warp_size;
  int batch_id = blockDim.y * blockIdx.x + threadIdx.y;

  // set effective_warp_id as 1 when warps do effective work,
  // when warps do ineffective work, effective_warp_id remains unchanged.
  int effective_warp_id = batch_size - batch_id;
  if (effective_warp_id > 1) effective_warp_id = 1;

  int thread_in_warp_idx = threadIdx.x;

  // 1.read data from global memory to registers
  AccT elements[warp_iter];
  // set effective_element_count as the num of elements when warps do effective
  // work
  // set effective_element_count as 0, when warps do ineffective work
  int effective_element_count = (effective_warp_id <= 0) ? 0 : element_count;
  for (int it = 0; it < warp_iter; ++it) {
    int element_index = thread_in_warp_idx + it * kernel_warp_size;
    if (element_index < effective_element_count) {
      elements[it] =
          static_cast<AccT>(src[batch_id * element_count + element_index]);
    } else {
      elements[it] = -std::numeric_limits<AccT>::infinity();
    }
  }

  // 2.compute max_value. For each thread, loop all registers to find max
  AccT max_value = elements[0];
#pragma unroll
  for (int it = 1; it < warp_iter; ++it) {
    max_value = (max_value > elements[it]) ? max_value : elements[it];
  }
  max_value = WarpReduceMax<AccT, kernel_warp_size>(max_value);

  // 3.For each warp, accumulate all thread registers
  AccT sum = 0.0f;
#pragma unroll
  for (int it = 0; it < warp_iter; ++it) {
    sum += std::exp(elements[it] - max_value);
  }
  sum = WarpReduceSum<AccT, kernel_warp_size>(sum);

  // 4.store result.
  sum = std::log(sum);
#pragma unroll
  for (int it = 0; it < warp_iter; ++it) {
    int element_index = thread_in_warp_idx + it * kernel_warp_size;
    if (element_index < element_count) {
      dst[batch_id * element_count + element_index] =
          static_cast<T>(elements[it] - max_value - sum);
    } else {
      break;
    }
  }
}

template <typename T, typename AccT>
void LaunchSoftmaxForwardForLastAxis(T *dst, const T *src, int dim_size,
                                     int outer_size, gpuStream_t stream) {
  int threads_per_block = 128;
  int near_greater_power_of_two = GetNearGreaterPowerOfTwo(dim_size);
  int kernel_warp_size =
      (near_greater_power_of_two < 32) ? near_greater_power_of_two : 32;
  int warps_per_block = (threads_per_block / kernel_warp_size);
  int blocks = (outer_size + warps_per_block - 1) / warps_per_block;
  dim3 threads(kernel_warp_size, warps_per_block, 1);

  switch (near_greater_power_of_two) {
    LAUNCH_WARP_FORWAR_COMPUTE(1);
    LAUNCH_WARP_FORWAR_COMPUTE(2);
    LAUNCH_WARP_FORWAR_COMPUTE(4);     // dim_size: 3~4
    LAUNCH_WARP_FORWAR_COMPUTE(8);     // dim_size: 5~8
    LAUNCH_WARP_FORWAR_COMPUTE(16);    // dim_size: 9~16
    LAUNCH_WARP_FORWAR_COMPUTE(32);    // dim_size: 17~32
    LAUNCH_WARP_FORWAR_COMPUTE(64);    // dim_size: 33~64
    LAUNCH_WARP_FORWAR_COMPUTE(128);   // dim_size 65~128
    LAUNCH_WARP_FORWAR_COMPUTE(256);   // dim_size 129~256
    LAUNCH_WARP_FORWAR_COMPUTE(512);   // dim_size 257~512
    LAUNCH_WARP_FORWAR_COMPUTE(1024);  // dim_size 513~1024

    default:
      break;
  }
}

template <typename T>
class LogSoftmaxKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
  using MPDType = typename details::MPTypeTrait<T>::Type;

 public:
  void Compute(const framework::ExecutionContext &context) const override {
    const auto *x = context.Input<framework::Tensor>("X");
    auto *out = context.Output<framework::Tensor>("Out");
    const auto *input_data = x->data<T>();
    auto *output_data = out->mutable_data<T>(context.GetPlace());

    const int rank = x->dims().size();
    const int axis = CanonicalAxis(context.Attr<int>("axis"), rank);

    int dim_size = x->dims()[axis];
    int inner_size = 1;
    for (int i = axis + 1; i < x->dims().size(); ++i) {
      inner_size *= x->dims()[i];
    }
    int outer_size = SizeToAxis(axis, x->dims());
    gpuStream_t stream = context.cuda_device_context().stream();

    if (inner_size == 1 && dim_size <= 1024 && dim_size * sizeof(T) <= 4096) {
      LaunchSoftmaxForwardForLastAxis<T, MPDType>(output_data, input_data,
                                                  dim_size, outer_size, stream);
    } else {
      LogSoftmaxFunctor<platform::CUDADeviceContext, T>()(
          context.template device_context<platform::CUDADeviceContext>(), x,
          out, axis);
    }
  }
};

}  // operators
}  // paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    log_softmax, ops::LogSoftmaxKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    log_softmax_grad, ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, float>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, double>,
    ops::LogSoftmaxGradKernel<plat::CUDADeviceContext, plat::float16>);
