/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/gather.cu.h"
#include "paddle/fluid/operators/gather_op.h"
#include "paddle/fluid/operators/scatter.cu.h"

namespace paddle {
namespace operators {

template <typename T>
class GatherOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *x = ctx.Input<Tensor>("X");
    auto *index = ctx.Input<Tensor>("Index");
    auto *output = ctx.Output<Tensor>("Out");

    output->mutable_data<T>(ctx.GetPlace());

    GPUGather<T>(ctx.device_context(), *x, *index, output);
  }
};

template <typename T>
class GatherGradOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *Index = ctx.Input<Tensor>("Index");
    auto *dX = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *dO = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto *x = ctx.Input<Tensor>("X");

    dX->mutable_data<T>(ctx.GetPlace());
    auto dxt = framework::EigenVector<T>::Flatten(*dX);
    auto &place = *ctx.template device_context<platform::CUDADeviceContext>()
                       .eigen_device();
    dxt.device(place) = dxt.constant(static_cast<T>(0));

    GPUScatterAssign<T>(ctx.device_context(), *dO, *Index, dX);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(gather, ops::GatherOpCUDAKernel<float>,
                        ops::GatherOpCUDAKernel<double>,
                        ops::GatherOpCUDAKernel<int64_t>,
                        ops::GatherOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(gather_grad, ops::GatherGradOpCUDAKernel<float>,
                        ops::GatherGradOpCUDAKernel<double>,
                        ops::GatherGradOpCUDAKernel<int64_t>,
                        ops::GatherGradOpCUDAKernel<int>);
