#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/operators/bincount_op.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/core/hostdevice.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using platform::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename InputT, typename OutT>
__global__ void KernelBincount(const InputT* input, const int total_elements,
                               const bool has_weights, const T* weights,
                               OutT* output) {
  if (!has_weights) {
    for (int i = threadIdx.x; i < total_elements; i += blockDim.x) {
      paddle::platform::CudaAtomicAdd(&output[input[i]], 1L);
    }
  } else {
    for (int i = threadIdx.x; i < total_elements; i += blockDim.x) {
      paddle::platform::CudaAtomicAdd(&output[input[i]],
                                      static_cast<OutT>(weights[i]));
    }
  }
}

template <typename DeviceContext, typename T, typename InputT>
void BincountCUDAInner(const framework::ExecutionContext& context) {
  const Tensor* input = context.Input<framework::Tensor>("X");
  const Tensor* weights = context.Input<framework::Tensor>("Weights");
  Tensor* output = context.Output<framework::Tensor>("Out");
  auto& minlength = context.Attr<int>("minlength");

  const InputT* input_data = input->data<InputT>();

  const int input_numel = input->numel();

  if (input_data == nullptr) {
    framework::DDim out_dim{0};
    output->Resize(out_dim);
    output->mutable_data<T>(context.GetPlace());
    return;
  }
  auto input_x = framework::EigenVector<InputT>::Flatten(*input);

  framework::Tensor input_min_t, input_max_t;
  auto* input_max_data =
      input_max_t.mutable_data<InputT>({1}, context.GetPlace());
  auto* input_min_data =
      input_min_t.mutable_data<InputT>({1}, context.GetPlace());

  auto input_max_scala = framework::EigenScalar<InputT>::From(input_max_t);
  auto input_min_scala = framework::EigenScalar<InputT>::From(input_min_t);

  auto* place = context.template device_context<DeviceContext>().eigen_device();
  input_max_scala.device(*place) = input_x.maximum();
  input_min_scala.device(*place) = input_x.minimum();

  Tensor input_min_cpu, input_max_cpu;
  paddle::framework::TensorCopySync(input_max_t, platform::CPUPlace(),
                                    &input_max_cpu);
  paddle::framework::TensorCopySync(input_min_t, platform::CPUPlace(),
                                    &input_min_cpu);

  InputT input_min = input_min_cpu.data<InputT>()[0];

  PADDLE_ENFORCE_GE(
      input_min, static_cast<InputT>(0),
      platform::errors::InvalidArgument(
          "The elements in input tensor must be non-negative ints"));

  int64_t output_size =
      static_cast<int64_t>(input_max_cpu.data<InputT>()[0]) + 1L;

  output_size = std::max(output_size, static_cast<int64_t>(minlength));
  framework::DDim out_dim{output_size};
  output->Resize(out_dim);

  bool has_weights = (weights != nullptr);

  const T* weights_data = has_weights ? weights->data<T>() : nullptr;

  auto stream =
      context.template device_context<platform::CUDADeviceContext>().stream();

  if (!has_weights) {
    int64_t* output_data = output->mutable_data<int64_t>(context.GetPlace());
    phi::funcs::SetConstant<DeviceContext, int64_t>()(
        context.template device_context<DeviceContext>(), output, 0L);

    KernelBincount<T, InputT, int64_t><<<GET_BLOCKS(input_numel),
                                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        input_data, input_numel, has_weights, weights_data, output_data);
  } else {
    const auto& weights_type = framework::TransToProtoVarType(weights->dtype());

    if (weights_type == framework::proto::VarType::FP32) {
      float* output_data = output->mutable_data<float>(context.GetPlace());
      phi::funcs::SetConstant<DeviceContext, float>()(
          context.template device_context<DeviceContext>(), output,
          static_cast<float>(0));

      KernelBincount<T, InputT, float><<<GET_BLOCKS(input_numel),
                                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          input_data, input_numel, has_weights, weights_data, output_data);
    } else {
      double* output_data = output->mutable_data<double>(context.GetPlace());
      phi::funcs::SetConstant<DeviceContext, double>()(
          context.template device_context<DeviceContext>(), output,
          static_cast<double>(0));

      KernelBincount<T, InputT, double><<<GET_BLOCKS(input_numel),
                                          PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          input_data, input_numel, has_weights, weights_data, output_data);
    }
  }
}

template <typename DeviceContext, typename T>
class BincountCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    const Tensor* input = context.Input<framework::Tensor>("X");
    const auto& input_type = framework::TransToProtoVarType(input->dtype());

    if (input_type == framework::proto::VarType::INT32) {
      BincountCUDAInner<DeviceContext, T, int>(context);
    } else if (input_type == framework::proto::VarType::INT64) {
      BincountCUDAInner<DeviceContext, T, int64_t>(context);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    bincount, ops::BincountCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::BincountCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>,
    ops::BincountCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::BincountCUDAKernel<paddle::platform::CUDADeviceContext, double>);
