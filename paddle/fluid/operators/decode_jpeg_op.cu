// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#if !defined(WITH_NV_JETSON) && !defined(PADDLE_WITH_HIP)

#include <string>
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/platform/dynload/nvjpeg.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/stream/hip_stream.h"

namespace paddle {
namespace operators {

static hipStream_t nvjpeg_stream = nullptr;
static nvjpegHandle_t nvjpeg_handle = nullptr;

void InitNvjpegImage(nvjpegImage_t* img) {
  for (int c = 0; c < NVJPEG_MAX_COMPONENT; c++) {
    img->channel[c] = nullptr;
    img->pitch[c] = 0;
  }
}

template <typename T>
class GPUDecodeJpegKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    // Create nvJPEG handle
    if (nvjpeg_handle == nullptr) {
      nvjpegStatus_t create_status =
          platform::dynload::nvjpegCreateSimple(&nvjpeg_handle);

      PADDLE_ENFORCE_EQ(create_status, NVJPEG_STATUS_SUCCESS,
                        platform::errors::Fatal("nvjpegCreateSimple failed: ",
                                                create_status));
    }

    nvjpegJpegState_t nvjpeg_state;
    nvjpegStatus_t state_status =
        platform::dynload::nvjpegJpegStateCreate(nvjpeg_handle, &nvjpeg_state);

    PADDLE_ENFORCE_EQ(state_status, NVJPEG_STATUS_SUCCESS,
                      platform::errors::Fatal("nvjpegJpegStateCreate failed: ",
                                              state_status));

    int components;
    nvjpegChromaSubsampling_t subsampling;
    int widths[NVJPEG_MAX_COMPONENT];
    int heights[NVJPEG_MAX_COMPONENT];

    auto* x = ctx.Input<framework::Tensor>("X");
    auto* x_data = x->data<T>();

    nvjpegStatus_t info_status = platform::dynload::nvjpegGetImageInfo(
        nvjpeg_handle, x_data, (size_t)x->numel(), &components, &subsampling,
        widths, heights);

    PADDLE_ENFORCE_EQ(
        info_status, NVJPEG_STATUS_SUCCESS,
        platform::errors::Fatal("nvjpegGetImageInfo failed: ", info_status));

    int width = widths[0];
    int height = heights[0];

    nvjpegOutputFormat_t output_format;
    int output_components;

    auto mode = ctx.Attr<std::string>("mode");
    if (mode == "unchanged") {
      if (components == 1) {
        output_format = NVJPEG_OUTPUT_Y;
        output_components = 1;
      } else if (components == 3) {
        output_format = NVJPEG_OUTPUT_RGB;
        output_components = 3;
      } else {
        platform::dynload::nvjpegJpegStateDestroy(nvjpeg_state);
        PADDLE_THROW(platform::errors::Fatal(
            "The provided mode is not supported for JPEG files on GPU"));
      }
    } else if (mode == "gray") {
      output_format = NVJPEG_OUTPUT_Y;
      output_components = 1;
    } else if (mode == "rgb") {
      output_format = NVJPEG_OUTPUT_RGB;
      output_components = 3;
    } else {
      platform::dynload::nvjpegJpegStateDestroy(nvjpeg_state);
      PADDLE_THROW(platform::errors::Fatal(
          "The provided mode is not supported for JPEG files on GPU"));
    }

    nvjpegImage_t out_image;
    InitNvjpegImage(&out_image);

    // create nvjpeg stream
    if (nvjpeg_stream == nullptr) {
      hipStreamCreateWithFlags(&nvjpeg_stream, hipStreamNonBlocking);
    }

    int sz = widths[0] * heights[0];

    auto* out = ctx.Output<framework::LoDTensor>("Out");
    std::vector<int64_t> out_shape = {output_components, height, width};
    out->Resize(framework::make_ddim(out_shape));

    T* data = out->mutable_data<T>(ctx.GetPlace());

    for (int c = 0; c < output_components; c++) {
      out_image.channel[c] = data + c * sz;
      out_image.pitch[c] = width;
    }

    nvjpegStatus_t decode_status = platform::dynload::nvjpegDecode(
        nvjpeg_handle, nvjpeg_state, x_data, x->numel(), output_format,
        &out_image, nvjpeg_stream);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(decode_jpeg, ops::GPUDecodeJpegKernel<uint8_t>)

#endif
