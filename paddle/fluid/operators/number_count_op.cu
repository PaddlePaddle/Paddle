#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
// The file has been adapted from the two files:
//     https://github.com/laekov/fastmoe/blob/master/cuda/local_exchange.cu
//     https://github.com/laekov/fastmoe/blob/master/cuda/local_exchange.cuh
//     Git commit hash: 295a615aacce7e54a37e7935274ba15e901c78e4
// We retain the following license from the original files:
//     Copyright 2021, Jiaao He. All rights reserved.
//  Licensed under the Apache License, Version 2.0 (the "License").

#include "paddle/fluid/operators/number_count_op.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

#define CEIL(_x_, _y_) (((_x_)-1) / (_y_) + 1)
#define PERTHREAD_EXPERTS 256
#define WARP_SIZE 32

const int CUDA_NUM_THREADS = 512;
static inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

using LoDTensor = framework::LoDTensor;
using Tensor = framework::Tensor;

template <typename T>
__global__ void initialize_zero_kernel(T* data, const int length) {
  CUDA_KERNEL_LOOP(idx, length) { data[idx] = static_cast<T>(0); }
}

template <typename T>
__global__ void NumberCount(const T* numbers,
                            T* number_count,
                            int64_t batch_size,
                            int upper_range) {
  int res_tmp[PERTHREAD_EXPERTS] = {0};
  int expert_min = blockIdx.x * PERTHREAD_EXPERTS;
  int expert_max = expert_min + PERTHREAD_EXPERTS;
  if (expert_max > upper_range) {
    expert_max = upper_range;
  }
  for (int i = threadIdx.x; i < batch_size; i += blockDim.x) {
    T idx = numbers[i];
    if (idx == -1) {
      continue;
    }
    if (idx < expert_min || idx >= expert_max) {
      continue;
    }
    res_tmp[idx - expert_min] += 1;
  }
  for (int i = expert_min; i < expert_max; ++i) {
    int x = res_tmp[i - expert_min];
#pragma unroll
    for (int j = 1; j < WARP_SIZE; j <<= 1) {
#ifdef __HIPCC__
      x = x + __shfl_down(x, j);
#else
      x = x + __shfl_down_sync(-1u, x, j);
#endif
    }
    if (threadIdx.x % WARP_SIZE == 0) {
      platform::CudaAtomicAdd(number_count + i, x);
    }
  }
}

template <typename T>
class NumberCountOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto numbers = context.Input<LoDTensor>("numbers");
    auto upper_range = context.Attr<int>("upper_range");
    auto number_count = context.Output<LoDTensor>("Out");

    int64_t batch_size = numbers->numel();
    auto place = context.GetPlace();
    const auto& dev_ctx = context.template device_context<phi::GPUContext>();

    framework::DDim out_dims = phi::make_ddim({upper_range});
    auto out_data = number_count->mutable_data<T>(out_dims, place);
    const T* gate_data = numbers->data<T>();

    initialize_zero_kernel<T>
        <<<GET_BLOCKS(upper_range), CUDA_NUM_THREADS, 0, dev_ctx.stream()>>>(
            out_data, upper_range);

    NumberCount<T>
        <<<CEIL(upper_range, PERTHREAD_EXPERTS), 256, 0, dev_ctx.stream()>>>(
            gate_data, out_data, batch_size, upper_range);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(number_count, ops::NumberCountOpCUDAKernel<int64_t>);
