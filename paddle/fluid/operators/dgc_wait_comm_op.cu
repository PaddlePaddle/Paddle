/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "dgc/dgc.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/fluid/operators/dgc_wait_comm_op.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/ProcessGroup.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif

namespace paddle {
namespace operators {
template <typename DeviceContext, typename T>
class DGCWaitCommOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto place = ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(place), true,
        platform::errors::PreconditionNotMet(
            "wait_comm op can run on gpu place only for now, but got %s",
            place.DebugString()));

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
    auto x = ctx.Input<framework::Tensor>("X");
    int rid = ctx.Attr<int>("ring_id");
    auto map = distributed::ProcessGroupMapFromGid::getInstance();
    PADDLE_ENFORCE_EQ(
    map->has(rid), true,
    platform::errors::InvalidArgument("dgc only nomally work after PaddlePaddle==2.3.1"));
    distributed::ProcessGroup* pg = map->get(rid);
    std::vector<phi::DenseTensor> in_tensor = {*x};
    std::vector<std::unique_ptr<phi::GPUContext>> ctxs = pg->GetDeviceContext(in_tensor);

    auto compute_stream =
        static_cast<phi::GPUContext*>(
            platform::DeviceContextPool::Instance().Get(place))
            ->stream();
    auto comm_stream = ctxs[0]->stream();

    gpuEvent_t event; 

    PADDLE_ENFORCE_GPU_SUCCESS(
        hipEventCreate(&event, hipEventDisableTiming));

#ifdef PADDLE_WITH_HIP
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventRecord(event, comm_stream));
    PADDLE_ENFORCE_GPU_SUCCESS(hipStreamWaitEvent(compute_stream, event, 0));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventRecord(event, comm_stream));
    PADDLE_ENFORCE_GPU_SUCCESS(hipStreamWaitEvent(compute_stream, event, 0));
#endif
#else
    PADDLE_THROW(platform::errors::PreconditionNotMet(
        "PaddlePaddle should compile with GPU."));
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    dgc_wait_comm,
    ops::DGCWaitCommOpCUDAKernel<phi::GPUContext, float>);
