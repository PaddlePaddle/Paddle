#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifndef PADDLE_WITH_HIP
// HIP not support cusolver
#include <algorithm>
#include <vector>
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_function.h"
#include "paddle/fluid/operators/matrix_rank_op.h"
#include "paddle/fluid/operators/svd_helper.h"
#include "paddle/fluid/platform/dynload/cusolver.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace paddle {
namespace operators {
namespace detail {
DDim GetUDDim(const DDim& x_dim, int k) {
  auto x_vec = phi::vectorize(x_dim);
  x_vec[x_vec.size() - 1] = k;
  return phi::make_ddim(x_vec);
}

DDim GetVHDDim(const DDim& x_dim, int k) {
  auto x_vec = phi::vectorize(x_dim);
  x_vec[x_vec.size() - 2] = k;
  return phi::make_ddim(x_vec);
}
}  // namespace detail

template <typename T>
class MatrixRankGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();

    const Tensor* x = context.Input<Tensor>("X");
    auto* x_data = x->data<T>();
    auto* out = context.Output<Tensor>("Out");
    out->mutable_data<int64_t>(context.GetPlace());
    bool hermitian = context.Attr<bool>("hermitian");

    auto dim_x = x->dims();
    auto dim_out = out->dims();
    int rows = dim_x[dim_x.size() - 2];
    int cols = dim_x[dim_x.size() - 1];
    int k = std::min(rows, cols);
    auto numel = x->numel();
    int batches = numel / (rows * cols);

    bool use_default_tol = context.Attr<bool>("use_default_tol");
    const Tensor* atol_tensor = nullptr;
    Tensor temp_tensor;
    T rtol_T = 0;
    if (use_default_tol) {
      framework::TensorFromVector<T>(std::vector<T>{0},
                                     context.device_context(), &temp_tensor);
      atol_tensor = &temp_tensor;
      rtol_T = std::numeric_limits<T>::epsilon() * std::max(rows, cols);
    } else if (context.HasInput("TolTensor")) {
      atol_tensor = context.Input<Tensor>("TolTensor");
    } else {
      framework::TensorFromVector<T>(std::vector<T>{context.Attr<float>("tol")},
                                     context.device_context(), &temp_tensor);
      atol_tensor = &temp_tensor;
    }

    // Must Copy X once, because the gesvdj will destory the content when exit.
    Tensor x_tmp;
    paddle::framework::TensorCopy(*x, context.GetPlace(), &x_tmp);
    auto info = memory::Alloc(dev_ctx, sizeof(int) * batches);
    int* info_ptr = reinterpret_cast<int*>(info->ptr());

    Tensor eigenvalue_tensor;
    auto* eigenvalue_data = eigenvalue_tensor.mutable_data<T>(
        detail::GetEigenvalueDim(dim_x, k), context.GetPlace());
    if (hermitian) {
      SyevjBatched(dev_ctx, batches, rows, x_tmp.data<T>(), eigenvalue_data,
                   info_ptr);
      platform::ForRange<platform::CUDADeviceContext> for_range(
          dev_ctx, eigenvalue_tensor.numel());
      phi::funcs::AbsFunctor<T> functor(eigenvalue_data, eigenvalue_data,
                                        eigenvalue_tensor.numel());
      for_range(functor);
    } else {
      Tensor U, VH;
      auto* u_data =
          U.mutable_data<T>(detail::GetUDDim(dim_x, k), context.GetPlace());
      auto* vh_data =
          VH.mutable_data<T>(detail::GetVHDDim(dim_x, k), context.GetPlace());
      GesvdjBatched(dev_ctx, batches, cols, rows, k, x_tmp.data<T>(), vh_data,
                    u_data, eigenvalue_data, info_ptr, 1);
    }

    auto dito_T =
        math::DeviceIndependenceTensorOperations<platform::CUDADeviceContext,
                                                 T>(context);
    std::vector<int> max_eigenvalue_shape =
        phi::vectorize<int>(detail::RemoveLastDim(eigenvalue_tensor.dims()));
    Tensor max_eigenvalue_tensor =
        dito_T.ReduceMax(eigenvalue_tensor, max_eigenvalue_shape);
    Tensor temp_rtol_tensor;
    framework::TensorFromVector<T>(std::vector<T>{rtol_T},
                                   context.device_context(), &temp_rtol_tensor);
    Tensor rtol_tensor = dito_T.Mul(temp_rtol_tensor, max_eigenvalue_tensor);
    Tensor tol_tensor;
    tol_tensor.mutable_data<T>(dim_out, context.GetPlace());
    ElementwiseComputeEx<GreaterElementFunctor<T>, platform::CUDADeviceContext,
                         T, T>(context, atol_tensor, &rtol_tensor, -1,
                               GreaterElementFunctor<T>(), &tol_tensor);

    tol_tensor.Resize(detail::NewAxisDim(tol_tensor.dims(), 1));

    Tensor compare_result;
    compare_result.mutable_data<int64_t>(detail::NewAxisDim(dim_out, k),
                                         context.GetPlace());
    int axis = -1;
    ElementwiseComputeEx<GreaterThanFunctor<T, int64_t>,
                         platform::CUDADeviceContext, T, int64_t>(
        context, &eigenvalue_tensor, &tol_tensor, axis,
        GreaterThanFunctor<T, int64_t>(), &compare_result);
    auto dito_int =
        math::DeviceIndependenceTensorOperations<platform::CUDADeviceContext,
                                                 int64_t>(context);
    std::vector<int> result_shape = phi::vectorize<int>(dim_out);
    Tensor result = dito_int.ReduceSum(compare_result, result_shape);
    out->ShareDataWith(result);
  }

  void GesvdjBatched(const platform::CUDADeviceContext& dev_ctx, int batchSize,
                     int m, int n, int k, T* A, T* U, T* V, T* S, int* info,
                     int thin_UV = 1) const;

  void SyevjBatched(const platform::CUDADeviceContext& dev_ctx, int batchSize,
                    int n, T* A, T* W, int* info) const;
};

template <>
void MatrixRankGPUKernel<float>::GesvdjBatched(
    const platform::CUDADeviceContext& dev_ctx, int batchSize, int m, int n,
    int k, float* A, float* U, float* V, float* S, int* info,
    int thin_UV) const {
  // do not compute singular vectors
  const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverGesvdjInfo_t gesvdj_params = NULL;
  int lda = m;
  int ldu = m;
  int ldt = n;
  int lwork = 0;
  auto handle = dev_ctx.cusolver_dn_handle();
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnSgesvdj_bufferSize(
      handle, jobz, thin_UV, m, n, A, lda, S, U, ldu, V, ldt, &lwork,
      gesvdj_params));
  auto workspace = memory::Alloc(dev_ctx, lwork * sizeof(float));
  float* workspace_ptr = reinterpret_cast<float*>(workspace->ptr());
  int stride_A = lda * n;
  int stride_U = ldu * (thin_UV ? k : m);
  int stride_V = ldt * (thin_UV ? k : n);
  for (int i = 0; i < batchSize; i++) {
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnSgesvdj(
        handle, jobz, thin_UV, m, n, A + stride_A * i, lda, S + k * i,
        U + stride_U * i, ldu, V + stride_V * i, ldt, workspace_ptr, lwork,
        info, gesvdj_params));
    int error_info;
    memory::Copy(platform::CPUPlace(), &error_info, dev_ctx.GetPlace(), info,
                 sizeof(int), dev_ctx.stream());
    PADDLE_ENFORCE_EQ(
        error_info, 0,
        platform::errors::PreconditionNotMet(
            "For batch [%d]: CUSolver SVD is not zero. [%d]", i, error_info));
  }
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

template <>
void MatrixRankGPUKernel<double>::GesvdjBatched(
    const platform::CUDADeviceContext& dev_ctx, int batchSize, int m, int n,
    int k, double* A, double* U, double* V, double* S, int* info,
    int thin_UV) const {
  // do not compute singular vectors
  const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverGesvdjInfo_t gesvdj_params = NULL;
  int lda = m;
  int ldu = m;
  int ldt = n;
  int lwork = 0;
  auto handle = dev_ctx.cusolver_dn_handle();
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnDgesvdj_bufferSize(
      handle, jobz, thin_UV, m, n, A, lda, S, U, ldu, V, ldt, &lwork,
      gesvdj_params));
  auto workspace = memory::Alloc(dev_ctx, lwork * sizeof(double));
  double* workspace_ptr = reinterpret_cast<double*>(workspace->ptr());
  int stride_A = lda * n;
  int stride_U = ldu * (thin_UV ? k : m);
  int stride_V = ldt * (thin_UV ? k : n);
  for (int i = 0; i < batchSize; ++i) {
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnDgesvdj(
        handle, jobz, thin_UV, m, n, A + stride_A * i, lda, S + k * i,
        U + stride_U * i, ldu, V + stride_V * i, ldt, workspace_ptr, lwork,
        info, gesvdj_params));
    // check the error info
    int error_info;
    memory::Copy(platform::CPUPlace(), &error_info, dev_ctx.GetPlace(), info,
                 sizeof(int), dev_ctx.stream());
    PADDLE_ENFORCE_EQ(
        error_info, 0,
        platform::errors::PreconditionNotMet(
            "For batch [%d]: CUSolver SVD is not zero. [%d]", i, error_info));
  }
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

template <>
void MatrixRankGPUKernel<float>::SyevjBatched(
    const platform::CUDADeviceContext& dev_ctx, int batchSize, int n, float* A,
    float* W, int* info) const {
  auto handle = dev_ctx.cusolver_dn_handle();
  // Compute eigenvalues only
  const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  // matrix is saved as column-major in cusolver.
  // numpy and torch use lower triangle to compute eigenvalues, so here use
  // upper triangle
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
  int lda = n;
  int stride_A = lda * n;
  int lwork = 0;
  hipsolverSyevjInfo_t params = NULL;
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnCreateSyevjInfo(&params));
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnSsyevj_bufferSize(
      handle, jobz, uplo, n, A, lda, W, &lwork, params));
  auto workspace = memory::Alloc(dev_ctx, lwork * sizeof(float));
  float* workspace_ptr = reinterpret_cast<float*>(workspace->ptr());
  for (int i = 0; i < batchSize; i++) {
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnSsyevj(
        handle, jobz, uplo, n, A + stride_A * i, lda, W + n * i, workspace_ptr,
        lwork, info, params));

    int error_info;
    memory::Copy(platform::CPUPlace(), &error_info, dev_ctx.GetPlace(), info,
                 sizeof(int), dev_ctx.stream());
    PADDLE_ENFORCE_EQ(
        error_info, 0,
        platform::errors::PreconditionNotMet(
            "For batch [%d]: CUSolver eigenvalues is not zero. [%d]", i,
            error_info));
  }
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnDestroySyevjInfo(params));
}

template <>
void MatrixRankGPUKernel<double>::SyevjBatched(
    const platform::CUDADeviceContext& dev_ctx, int batchSize, int n, double* A,
    double* W, int* info) const {
  auto handle = dev_ctx.cusolver_dn_handle();
  // Compute eigenvalues only
  const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  //  upper triangle of A is stored
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
  int lda = n;
  int stride_A = lda * n;
  int lwork = 0;
  hipsolverSyevjInfo_t params = NULL;
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnCreateSyevjInfo(&params));
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnDsyevj_bufferSize(
      handle, jobz, uplo, n, A, lda, W, &lwork, params));
  auto workspace = memory::Alloc(dev_ctx, lwork * sizeof(double));
  double* workspace_ptr = reinterpret_cast<double*>(workspace->ptr());

  for (int i = 0; i < batchSize; i++) {
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::hipsolverDnDsyevj(
        handle, jobz, uplo, n, A + stride_A * i, lda, W + n * i, workspace_ptr,
        lwork, info, params));
    int error_info;
    memory::Copy(platform::CPUPlace(), &error_info, dev_ctx.GetPlace(), info,
                 sizeof(int), dev_ctx.stream());
    PADDLE_ENFORCE_EQ(
        error_info, 0,
        platform::errors::PreconditionNotMet(
            "For batch [%d]: CUSolver eigenvalues is not zero. [%d]", i,
            error_info));
  }
  PADDLE_ENFORCE_GPU_SUCCESS(
      platform::dynload::hipsolverDnDestroySyevjInfo(params));
}

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(matrix_rank, ops::MatrixRankGPUKernel<float>,
                        ops::MatrixRankGPUKernel<double>);
#endif  // not PADDLE_WITH_HIP
