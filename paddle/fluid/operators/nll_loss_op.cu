#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <algorithm>
#include <functional>
#include <string>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/operators/math.h"
#include "paddle/fluid/operators/nll_loss_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/hostdevice.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;
static const int NTHREADS = 32;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void GPUNLLLossForward1D_no_reduce(T* out_data, const T* x_data,
                                              const int64_t* label_data,
                                              const T* weight_data,
                                              const int64_t batch_size,
                                              const int64_t n_classes,
                                              const int64_t ignore_index) {
  CUDA_1D_KERNEL_LOOP(i, batch_size) {
    const int64_t cur_label = label_data[i];
    if (cur_label == ignore_index) {
      out_data[i] = 0;
      continue;
    }
    const T cur_weight = weight_data ? weight_data[cur_label] : (T)1;
    out_data[i] = -x_data[i * n_classes + cur_label] * cur_weight;
  }
}

template <typename T>
__global__ void GPUNLLLossForward1D_with_reduce(
    T* out_data, T* total_weight_data, const T* x_data,
    const int64_t* label_data, const T* weight_data, const int64_t batch_size,
    const int64_t n_classes, const int64_t size_average,
    const int64_t ignore_index) {
  __shared__ T sharedInputs[NTHREADS], sharedWeights[NTHREADS];
  sharedInputs[threadIdx.x] = 0;
  sharedWeights[threadIdx.x] = 0;
  int i;
  for (i = threadIdx.x; i < batch_size; i += NTHREADS) {
    const auto cur_label = label_data[i];
    if (cur_label != ignore_index) {
      const auto cur_weight = weight_data ? weight_data[cur_label] : (T)1;
      sharedInputs[threadIdx.x] -=
          x_data[i * n_classes + cur_label] * cur_weight;
      sharedWeights[threadIdx.x] += cur_weight;
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    *out_data = *total_weight_data = 0;
    T output_val = 0;
    T total_weight_val = 0;
    for (i = 0; i < NTHREADS; ++i) {
      output_val += sharedInputs[i];
      total_weight_val += sharedWeights[i];
    }
    *total_weight_data = total_weight_val;
    *out_data = output_val;

    if (size_average && *total_weight_data != 0) {
      *out_data = output_val / total_weight_val;
    }
  }
}

// Reduce N values concurrently, i.e. suppose N = 2, and there are 4 threads:
// (1, 2), (3, 4), (5, 6), (7, 8), then the return in threadVals for thread 0
// is (1 + 3 + 5 + 7, 2 + 4 + 6 + 8) = (16, 20)
//
// If smem is not used again, there is no need to __syncthreads before this
// call. However, if smem will be used, e.g., this function is called in a loop,
// then __syncthreads is needed either before or afterwards to prevent non-0
// threads overriding smem in the next loop before num-0 thread reads from it.
template <typename T, typename ReduceOp, int N>
__device__ void reduceNValuesInBlock(T* smem, T threadVals[N],
                                     const unsigned int numVals,
                                     ReduceOp reduceOp, T init) {
  if (numVals == 0) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      threadVals[i] = init;
    }
    return;
  }

  // We store each of the N values contiguously, so if N = 2, all values for
  // the first threadVal for each thread in the block are stored followed by
  // all of the values for the second threadVal for each thread in the block
  if (threadIdx.x < numVals) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      smem[i * numVals + threadIdx.x] = threadVals[i];
    }
  }
  __syncthreads();

  // Number of lanes in the final reduction --> this is used to determine
  // where to put the outputs of each of the n things we are reducing. If
  // nLP = 32, then we have the 32 outputs for the first threadVal,
  // followed by the 32 outputs for the second threadVal, etc.
  const unsigned int numLanesParticipating = min(numVals, warpSize);

  if (numVals > warpSize && ((threadIdx.x / warpSize) == 0)) {
#pragma unroll
    for (int i = 0; i < N; ++i) {
      threadVals[i] = threadIdx.x < numVals ? threadVals[i] : init;
    }

    for (int i = warpSize + threadIdx.x; i < numVals; i += warpSize) {
#pragma unroll
      for (int j = 0; j < N; ++j) {
        threadVals[j] = reduceOp(threadVals[j], smem[j * numVals + i]);
      }
    }

#pragma unroll
    for (int i = 0; i < N; ++i) {
      smem[i * numLanesParticipating + threadIdx.x] = threadVals[i];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    if (numLanesParticipating == 32) {
#pragma unroll
      for (int i = 0; i < N; ++i) {
#pragma unroll
        for (int j = 1; j < 32; ++j) {
          threadVals[i] = reduceOp(threadVals[i], smem[i * 32 + j]);
        }
      }
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) {
        for (int j = 1; j < numLanesParticipating; ++j) {
          threadVals[i] = reduceOp(threadVals[i], smem[i * numVals + j]);
        }
      }
    }
  }
}

// Block-wide reduction in shared memory helper; only threadIdx.x == 0 will
// return the reduced value
//
// If smem is not used again, there is no need to __syncthreads before this
// call. However, if smem will be used, e.g., this function is called in a loop,
// then __syncthreads is needed either before or afterwards to prevent non-0
// threads overriding smem in the next loop before num-0 thread reads from it.
template <typename T, typename ReduceOp>
__device__ T reduceBlock(T* smem, const unsigned int numVals, T threadVal,
                         ReduceOp reduceOp, T init) {
  reduceNValuesInBlock<T, ReduceOp, 1>(smem, &threadVal, numVals, reduceOp,
                                       init);
  return threadVal;
}

template <typename T>
__global__ void GPUNLLLossForward2D_no_reduce(
    T* out_data, const T* x_data, const int64_t* label_data,
    const T* weight_data, const int64_t batch_size, const int64_t n_classes,
    const int64_t in_dim2, const int64_t in_dim3, const int64_t ignore_index) {
  const int64_t map_size = in_dim2 * in_dim3;
  const int64_t sample_size = n_classes * map_size;
  const int64_t out_numel = batch_size * map_size;
  CUDA_1D_KERNEL_LOOP(i, out_numel) {
    const int64_t b = i % batch_size;
    const int64_t h = (i / batch_size) % in_dim2;
    const int64_t w = (i / (batch_size * in_dim2)) % in_dim3;

    const int64_t index = b * map_size + h * in_dim3 + w;
    const int64_t cur_label = label_data[index];
    if (cur_label == ignore_index) {
      out_data[index] = 0;
      continue;
    }
    const T cur_weight = weight_data ? weight_data[cur_label] : (T)1;
    out_data[index] =
        -x_data[b * sample_size + cur_label * map_size + h * in_dim3 + w] *
        cur_weight;
  }
}

template <typename T>
__global__ void GPUNLLLossForward2D_with_reduce(
    T* out_data, T* total_weight_data, const T* x_data,
    const int64_t* label_data, const T* weight_data, const int64_t batch_size,
    const int64_t n_classes, const int64_t map_nelem,
    const int64_t blocks_per_sample, const int64_t ignore_index) {
  __shared__ T partial_sums[kNumCUDAThreads];
  int64_t i;
  T input_sum = 0;
  T acc_weight = 0;
  *out_data = 0;
  *total_weight_data = 0;

  int64_t sample = blockIdx.x / blocks_per_sample;
  int64_t toffset = sample * map_nelem;
  int64_t ioffset = sample * map_nelem * n_classes;
  int64_t step = blockDim.x * blocks_per_sample;
  for (i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
       i < map_nelem; i += step) {
    const int64_t cur_label = label_data[toffset + i];
    if (cur_label != ignore_index) {
      const T cur_weight = weight_data ? weight_data[cur_label] : (T)1;
      input_sum -= x_data[ioffset + i + map_nelem * cur_label] * cur_weight;
      acc_weight += cur_weight;
    }
  }

  input_sum =
      reduceBlock(partial_sums, blockDim.x, input_sum, thrust::plus<T>(), (T)0);
  __syncthreads();
  acc_weight = reduceBlock(partial_sums, blockDim.x, acc_weight,
                           thrust::plus<T>(), (T)0);

  if (threadIdx.x == 0) {
    paddle::platform::CudaAtomicAdd(total_weight_data, acc_weight);
    paddle::platform::CudaAtomicAdd(out_data, input_sum);
  }
}

template <typename T>
__global__ void GPUNLLLossForward2D_size_average(T* out_data,
                                                 T* total_weight_data) {
  if (*total_weight_data != 0) {
    *out_data /= *total_weight_data;
  }
}

template <typename T>
__global__ void GPUNLLLossBackward1D_no_reduce(
    T* dx_data, const int64_t* label_data, const T* weight_data,
    const T* dout_data, const int64_t batch_size, const int64_t n_classes,
    const int64_t ignore_index) {
  CUDA_1D_KERNEL_LOOP(i, batch_size) {
    const int64_t cur_label = label_data[i];
    if (cur_label == ignore_index) {
      continue;
    }
    const T cur_weight = weight_data ? weight_data[cur_label] : (T)1;
    dx_data[i * n_classes + cur_label] = -dout_data[i] * cur_weight;
  }
}

template <typename T>
__global__ void GPUNLLLossBackward1D_with_reduce(
    T* dx_data, const T* total_weight_data, const int64_t* label_data,
    const T* weight_data, const T* dout_data, const int64_t batch_size,
    const int64_t n_classes, const int64_t size_average,
    const int64_t ignore_index) {
  if (*total_weight_data <= 0) {
    return;
  }
  int i;
  const T norm = size_average ? (T)(1 / *total_weight_data) : (T)1;
  for (i = threadIdx.x; i < batch_size; i += NTHREADS) {
    const int64_t cur_label = label_data[i];
    if (cur_label != ignore_index) {
      const T cur_weight = weight_data ? weight_data[cur_label] : (T)1;
      dx_data[i * n_classes + cur_label] = -cur_weight * dout_data[0] * norm;
    }
  }
}

template <typename T>
__global__ void GPUNLLLossBackward2D_no_reduce(
    T* dx_data, const int64_t* label_data, const T* weight_data,
    const T* dout_data, const int64_t batch_size, const int64_t n_classes,
    const int64_t in_dim2, const int64_t in_dim3, const int64_t ignore_index) {
  const int64_t map_size = in_dim2 * in_dim3;
  const int64_t sample_size = n_classes * map_size;
  const int64_t out_numel = batch_size * map_size;
  CUDA_1D_KERNEL_LOOP(i, out_numel) {
    const int64_t b = i % batch_size;
    const int64_t h = (i / batch_size) % in_dim2;
    const int64_t w = (i / (batch_size * in_dim2)) % in_dim3;
    const int64_t index = b * map_size + h * in_dim3 + w;
    const int64_t cur_label = label_data[index];
    if (cur_label == ignore_index) {
      continue;
    }
    const T cur_weight = weight_data ? weight_data[cur_label] : (T)1;
    dx_data[b * sample_size + cur_label * map_size + h * in_dim3 + w] =
        -dout_data[index] * cur_weight;
  }
}

template <typename T>
__global__ void GPUNLLLossBackward2D_with_reduce(
    T* dx_data, const T* total_weight_data, const int64_t* label_data,
    const T* weight_data, const T* dout_data, const int64_t batch_size,
    const int64_t n_classes, const int64_t map_nelem,
    const int64_t blocks_per_sample, const int64_t size_average,
    const int64_t ignore_index) {
  if (*total_weight_data <= 0) {
    return;
  }
  int64_t i;
  const T norm = size_average ? (T)(1 / *total_weight_data) : (T)1;
  int sample = blockIdx.x / blocks_per_sample;
  int step = blockDim.x * blocks_per_sample;
  int toffset = sample * map_nelem;
  int ioffset = sample * map_nelem * n_classes;
  for (i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
       i < map_nelem; i += step) {
    const int64_t cur_label = label_data[toffset + i];
    if (cur_label != ignore_index) {
      dx_data[ioffset + i + map_nelem * cur_label] =
          -(weight_data ? weight_data[cur_label] : (T)1) * norm * dout_data[0];
    }
  }
}

template <typename DeviceContext, typename T>
class NLLLossCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* labels = ctx.Input<Tensor>("Label");
    auto* weight = ctx.Input<Tensor>("Weight");
    auto* out = ctx.Output<Tensor>("Out");
    auto* total_weight = ctx.Output<Tensor>("Total_weight");
    auto ignore_index = ctx.Attr<int64_t>("ignore_index");
    auto reduction = ctx.Attr<std::string>("reduction");

    auto x_data = x->data<T>();
    auto out_data = out->mutable_data<T>(ctx.GetPlace());
    auto total_weight_data = total_weight->mutable_data<T>(ctx.GetPlace());
    auto label_data = labels->data<int64_t>();
    auto weight_data = weight ? weight->data<T>() : nullptr;
    hipMemset(total_weight_data, 0, sizeof(T));
    auto x_dims = x->dims();
    auto batch_size = x_dims[0];
    auto n_classes = x_dims[1];
    int64_t size_average = (int64_t)(reduction == "mean");

    if (x_dims.size() == 2) {
      int blocks = NumBlocks(batch_size);
      int threads = kNumCUDAThreads;
      auto& dev_ctx = ctx.cuda_device_context();
      if (reduction == "none") {
        GPUNLLLossForward1D_no_reduce<
            T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            out_data, x_data, label_data, weight_data, batch_size, n_classes,
            ignore_index);
      } else {
        GPUNLLLossForward1D_with_reduce<
            T><<<1, NTHREADS, 0, dev_ctx.stream()>>>(
            out_data, total_weight_data, x_data, label_data, weight_data,
            batch_size, n_classes, size_average, ignore_index);
      }
    } else if (x_dims.size() == 4) {
      const auto in_dim2 = x_dims[2];
      const auto in_dim3 = x_dims[3];
      const auto map_size = in_dim2 * in_dim3;
      const auto out_numel = batch_size * in_dim2 * in_dim3;
      int blocks = NumBlocks(out_numel);
      int threads = kNumCUDAThreads;
      auto& dev_ctx = ctx.cuda_device_context();
      if (reduction == "none") {
        GPUNLLLossForward2D_no_reduce<
            T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            out_data, x_data, label_data, weight_data, batch_size, n_classes,
            in_dim2, in_dim3, ignore_index);
      } else {
        int blocks_per_sample = NumBlocks(map_size) / 128;
        blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
        int total_blocks = blocks_per_sample * batch_size;
        GPUNLLLossForward2D_with_reduce<
            T><<<total_blocks, threads, 0, dev_ctx.stream()>>>(
            out_data, total_weight_data, x_data, label_data, weight_data,
            batch_size, n_classes, map_size, blocks_per_sample, ignore_index);
        if (size_average) {
          GPUNLLLossForward2D_size_average<T><<<1, 1, 0, dev_ctx.stream()>>>(
              out_data, total_weight_data);
        }
      }
    }
  }
};

template <typename DeviceContext, typename T>
class NLLLossGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* labels = ctx.Input<Tensor>("Label");
    auto* weight = ctx.Input<Tensor>("Weight");
    auto* total_weight = ctx.Input<Tensor>("Total_weight");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto dx_data = dx->mutable_data<T>(ctx.GetPlace());
    auto dout_data = dout->data<T>();
    auto label_data = labels->data<int64_t>();
    auto weight_data = weight ? weight->data<T>() : nullptr;
    auto total_weight_data = total_weight->data<T>();
    auto ignore_index = ctx.Attr<int64_t>("ignore_index");
    auto reduction = ctx.Attr<std::string>("reduction");
    hipMemset(dx_data, 0, dx->numel() * sizeof(T));

    int64_t size_average = (int64_t)(reduction == "mean");
    auto x_dims = x->dims();
    auto batch_size = x_dims[0];
    auto n_classes = x_dims[1];

    if (x_dims.size() == 2) {
      int blocks = NumBlocks(batch_size);
      int threads = kNumCUDAThreads;
      auto& dev_ctx = ctx.cuda_device_context();
      if (reduction == "none") {
        GPUNLLLossBackward1D_no_reduce<
            T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            dx_data, label_data, weight_data, dout_data, batch_size, n_classes,
            ignore_index);
      } else {
        GPUNLLLossBackward1D_with_reduce<
            T><<<1, NTHREADS, 0, dev_ctx.stream()>>>(
            dx_data, total_weight_data, label_data, weight_data, dout_data,
            batch_size, n_classes, size_average, ignore_index);
      }
    } else if (x_dims.size() == 4) {
      const auto in_dim2 = x_dims[2];
      const auto in_dim3 = x_dims[3];
      const auto map_size = in_dim2 * in_dim3;
      const auto out_numel = batch_size * in_dim2 * in_dim3;

      int blocks = NumBlocks(out_numel);
      int threads = kNumCUDAThreads;
      auto& dev_ctx = ctx.cuda_device_context();
      if (reduction == "none") {
        GPUNLLLossBackward2D_no_reduce<
            T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            dx_data, label_data, weight_data, dout_data, batch_size, n_classes,
            in_dim2, in_dim3, ignore_index);
      } else {
        int blocks_per_sample = NumBlocks(map_size) / 128;
        blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
        int total_blocks = blocks_per_sample * batch_size;
        GPUNLLLossBackward2D_with_reduce<
            T><<<total_blocks, threads, 0, dev_ctx.stream()>>>(
            dx_data, total_weight_data, label_data, weight_data, dout_data,
            batch_size, n_classes, map_size, blocks_per_sample, size_average,
            ignore_index);
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    nll_loss,
    ops::NLLLossCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::NLLLossCUDAKernel<paddle::platform::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    nll_loss_grad,
    ops::NLLLossGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::NLLLossGradCUDAKernel<paddle::platform::CUDADeviceContext, double>);
