#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/nccl_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;

template <typename T, int BlockDim, framework::DataLayout layout>
__global__ void KeLocalStats(const T *x, int N, int M, int C, T *mean_var) {
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int k = blockIdx.x; k < C; k += gridDim.x) {
    T x_sum = 0;
    T x2_sum = 0;
    for (int i = threadIdx.x; i < N * M; i += BlockDim) {
      int id = layout == framework::DataLayout::kNCHW
                   ? (i / M) * C * M + k * M + i % M
                   : i * C + k;
      T x_in = x[id];
      x_sum += x_in;
      x2_sum += x_in * x_in;
    }
    __syncthreads();
    T out = BlockReduce(temp_storage).Reduce(x_sum, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      mean_var[k] = out / (N * M);
    }
    out = BlockReduce(temp_storage).Reduce(x2_sum, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      mean_var[k + C] = out / (N * M);
    }
  }
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    mean_var[2 * C] = static_cast<T>(1.0);
  }
}

template <typename T>
__global__ void KeSyncAndMovingStats(T *means, T *variances, T *num_dev,
                                     const int C, const T momentum,
                                     const double epsilon, T *sv_mean_data,
                                     T *sv_inv_var_data, T *moving_means,
                                     T *moving_variances) {
  // sync stats across multi-devices
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < C; i += stride) {
    T mean = means[i] / (*num_dev);
    T var = variances[i] / (*num_dev);
    var = var - mean * mean;

    // sync stats
    sv_mean_data[i] = mean;
    sv_inv_var_data[i] = 1.0 / sqrt(var + epsilon);
    variances[i] = var;

    // moving stats
    moving_means[i] = moving_means[i] * momentum + mean * (1. - momentum);
    moving_variances[i] =
        moving_variances[i] * momentum + var * (1. - momentum);
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeNormAffine(const T *x, const T *scale, const T *bias,
                                    const T *mean, const T *variance,
                                    const double epsilon, const int C,
                                    const int M, const int num, T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? (i / M) % C : i % C;
    y[i] = (x[i] - mean[c]) / sqrt(variance[c] + epsilon) * scale[c] + bias[c];
  }
}

template <typename DeviceContext, typename T>
class SyncBatchNormKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const std::string layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout layout = framework::StringToDataLayout(layout_str);
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    PADDLE_ENFORCE(
        !use_global_stats,
        "sync_batch_norm doesn't support to set use_global_stats True. ",
        "Please use batch_norm in this case.");

    const auto *x = ctx.Input<Tensor>("X");
    const auto &x_dims = x->dims();
    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, layout, &N, &C, &H, &W, &D);
    int x_numel = x->numel();

    const T *x_d = x->data<T>();
    const T *s_d = ctx.Input<Tensor>("Scale")->data<T>();
    const T *b_d = ctx.Input<Tensor>("Bias")->data<T>();

    auto *y = ctx.Output<Tensor>("Y");
    T *y_d = y->mutable_data<T>(ctx.GetPlace());

    const T *mean_data = nullptr;
    const T *var_data = nullptr;

    auto &dev_ctx = ctx.cuda_device_context();
    auto stream = dev_ctx.stream();
    auto *comm = dev_ctx.nccl_comm();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();

    paddle::memory::AllocationPtr alloc_ptr{nullptr};

    if (is_test) {
      const auto *est_mean = ctx.Input<Tensor>("Mean");
      const auto *est_var = ctx.Input<Tensor>("Variance");
      mean_data = est_mean->data<T>();
      var_data = est_var->data<T>();
    } else {
      auto &allocator =
          platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx);
      // x, x^2, 1, here 1 is used to calc device num
      // device num also can be got from platform::DeviceContextPool
      const int bytes = (C * 2 + 1) * sizeof(T);
      alloc_ptr = allocator.Allocate(bytes);

      T *stats = reinterpret_cast<T *>(alloc_ptr->ptr());
      const int threads = 256;
      int grid = std::min(C, (max_threads + threads - 1) / threads);
      if (layout == framework::DataLayout::kNCHW) {
        KeLocalStats<
            T, threads,
            framework::DataLayout::kNCHW><<<grid, threads, 0, stream>>>(
            x_d, N, H * W * D, C, stats);
      } else {
        KeLocalStats<
            T, threads,
            framework::DataLayout::kNHWC><<<grid, threads, 0, stream>>>(
            x_d, N, H * W * D, C, stats);
      }

      Tensor c_g_st;
      T *c_g_st_d = c_g_st.mutable_data<T>({2 * C + 1}, platform::CPUPlace());
      auto gplace = boost::get<platform::CUDAPlace>(ctx.GetPlace());
      memory::Copy(platform::CPUPlace(), c_g_st_d, gplace, stats, bytes, 0);

      int dtype = platform::ToNCCLDataType(x->type());
      // In-place operation
      PADDLE_ENFORCE(platform::dynload::ncclAllReduce(
          stats, stats, 2 * C + 1, static_cast<ncclDataType_t>(dtype), ncclSum,
          comm, stream));

      // moving mean/variance
      auto *mean_out = ctx.Output<Tensor>("MeanOut");
      auto *variance_out = ctx.Output<Tensor>("VarianceOut");
      T *est_mean_data = mean_out->mutable_data<T>(ctx.GetPlace());
      T *est_var_data = variance_out->mutable_data<T>(ctx.GetPlace());

      auto *saved_mean = ctx.Output<Tensor>("SavedMean");
      auto *saved_inv_variance = ctx.Output<Tensor>("SavedVariance");
      T *sv_mean_data = saved_mean->mutable_data<T>(ctx.GetPlace());
      T *sv_inv_var_data = saved_inv_variance->mutable_data<T>(ctx.GetPlace());

      // Note, Input('Mean')/Input('Variance') share variable with
      // Output('MeanOut')/Output('VarianceOut')
      KeSyncAndMovingStats<T><<<(C + block - 1) / block, block, 0, stream>>>(
          stats, stats + C, stats + 2 * C, C, momentum, epsilon, sv_mean_data,
          sv_inv_var_data, est_mean_data, est_var_data);

      mean_data = sv_mean_data;
      var_data = stats + C;
    }

    int grid2 = (std::min(x_numel, max_threads) + block - 1) / block;
    if (layout == framework::DataLayout::kNCHW) {
      KeNormAffine<T,
                   framework::DataLayout::kNCHW><<<grid2, block, 0, stream>>>(
          x_d, s_d, b_d, mean_data, var_data, epsilon, C, H * W * D, x_numel,
          y_d);
    } else {
      KeNormAffine<T,
                   framework::DataLayout::kNHWC><<<grid2, block, 0, stream>>>(
          x_d, s_d, b_d, mean_data, var_data, epsilon, C, H * W * D, x_numel,
          y_d);
    }
  }
};

template <typename T, const int BlockDim, framework::DataLayout layout>
__global__ void KeBackwardLocalStats(const T *dy, const T *x, const T *means,
                                     int N, int M, int C, T *sum_dy_prod) {
  typedef hipcub::BlockReduce<double, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int k = blockIdx.x; k < C; k += gridDim.x) {
    T sum1 = 0;
    T sum2 = 0;
    T mean = means[k];
    for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
      int id = layout == framework::DataLayout::kNCHW
                   ? (i / M) * C * M + k * M + i % M
                   : i * C + k;
      T g = dy[id];
      sum1 += g;
      sum2 += g * (x[id] - mean);
    }

    __syncthreads();
    T out = BlockReduce(temp_storage).Reduce(sum1, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      sum_dy_prod[k] = out;
    }
    out = BlockReduce(temp_storage).Reduce(sum2, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      sum_dy_prod[k + C] = out;
    }
  }
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    sum_dy_prod[2 * C] = static_cast<T>(1.0);
  }
}

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void KeBNBackwardScaleBias(const T *dy, const T *x,
                                             const T *mean,
                                             const T *inv_variance,
                                             const double epsilon, const int N,
                                             const int C, const int HxW,
                                             T *dscale, T *dbias) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<double, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    T ds_sum = static_cast<T>(0);
    T db_sum = static_cast<T>(0);

    T inv_var_i = inv_variance[i];
    T mean_i = mean[i];
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int id = layout == framework::DataLayout::kNCHW
                         ? ((j / HxW) * C + i) * HxW + (j % HxW)
                         : j * outer_size + i;
      ds_sum += dy[id] * (x[id] - mean_i);
      db_sum += dy[id];
    }
    __syncthreads();
    double os = BlockReduce(temp_storage)
                    .Reduce(static_cast<double>(ds_sum), hipcub::Sum());
    __syncthreads();
    double ob = BlockReduce(temp_storage)
                    .Reduce(static_cast<double>(db_sum), hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      dscale[i] = static_cast<T>(os * inv_var_i);
      dbias[i] = static_cast<T>(ob);
    }
    __syncthreads();
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeBNBackwardData(const T *dy, const T *x, const T *beta,
                                        const T *mean, const T *inv_variance,
                                        const T *g_sum_dy,
                                        const T *g_sum_dy_prod,
                                        const T *num_dev, const double epsilon,
                                        const int C, const int HxW,
                                        const int num, T *dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  T scale = static_cast<T>(C) / num;
  T dev_num = num_dev[0];
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? i / HxW % C : i % C;
    T inv_var = inv_variance[c];
    T s_d = beta[c];
    T gvar = -1.0 * (g_sum_dy_prod[c] / dev_num) * s_d * inv_var *
             (inv_var * inv_var);
    T gmean = -1.0 * (g_sum_dy[c] / dev_num) * s_d * inv_var;

    dx[i] =
        dy[i] * s_d * inv_var + gmean * scale + gvar * scale * (x[i] - mean[c]);
  }
}

// Deriving the Gradient for the Backward Pass of Batch Normalization
// https://kevinzakka.github.io/2016/09/14/batch_normalization/
template <typename DeviceContext, typename T>
class SyncBatchNormGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const std::string layout_str = ctx.Attr<std::string>("data_layout");

    const DataLayout layout = framework::StringToDataLayout(layout_str);
    const auto *x = ctx.Input<Tensor>("X");
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");

    const auto &x_dims = x->dims();

    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, layout, &N, &C, &H, &W, &D);

    // init output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    d_x->mutable_data<T>(ctx.GetPlace());
    if (d_scale && d_bias) {
      d_scale->mutable_data<T>(ctx.GetPlace());
      d_bias->mutable_data<T>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(scale->dims().size(), 1UL);
    PADDLE_ENFORCE_EQ(scale->dims()[0], C);

    std::vector<int> dims;
    std::vector<int> strides;
    if (layout == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * C * D, 1, W * D * C, D * C, C};
    }

    const T *x_d = x->data<T>();
    const T *dy_d = d_y->data<T>();

    auto &dev_ctx = ctx.cuda_device_context();
    auto stream = dev_ctx.stream();
    auto *comm = dev_ctx.nccl_comm();

    const T *saved_mean = ctx.Input<Tensor>("SavedMean")->data<T>();
    const T *saved_inv_var = ctx.Input<Tensor>("SavedVariance")->data<T>();
    auto &allocator =
        platform::DeviceTemporaryAllocator::Instance().Get(dev_ctx);
    const int bytes = (C * 2 + 1) * sizeof(T);
    auto alloc_ptr = allocator.Allocate(bytes);
    T *stats = reinterpret_cast<T *>(alloc_ptr->ptr());

    const int threads = 256;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    int grid = std::min(C, (max_threads + threads - 1) / threads);
    int x_numel = x->numel();
    int fsize = H * W * D;

    if (layout == framework::DataLayout::kNCHW) {
      KeBackwardLocalStats<
          T, threads,
          framework::DataLayout::kNCHW><<<grid, threads, 0, stream>>>(
          dy_d, x_d, saved_mean, N, fsize, C, stats);
    } else {
      KeBackwardLocalStats<
          T, threads,
          framework::DataLayout::kNHWC><<<grid, threads, 0, stream>>>(
          dy_d, x_d, saved_mean, N, fsize, C, stats);
    }
    int dtype = platform::ToNCCLDataType(x->type());
    // In-place operation
    PADDLE_ENFORCE(platform::dynload::ncclAllReduce(
        stats, stats, 2 * C + 1, static_cast<ncclDataType_t>(dtype), ncclSum,
        comm, stream));

    const int block = 512;
    int grid2 = (std::min(x_numel, max_threads) + block - 1) / block;
    if (layout == framework::DataLayout::kNCHW) {
      if (d_scale && d_bias) {
        KeBNBackwardScaleBias<
            T, threads,
            framework::DataLayout::kNCHW><<<grid, threads, 0, stream>>>(
            dy_d, x_d, saved_mean, saved_inv_var, epsilon, N, C, fsize,
            d_scale->data<T>(), d_bias->data<T>());
      }
      if (d_x) {
        KeBNBackwardData<
            T, framework::DataLayout::kNCHW><<<grid2, block, 0, stream>>>(
            dy_d, x_d, scale->data<T>(), saved_mean, saved_inv_var, stats,
            stats + C, stats + 2 * C, epsilon, C, fsize, x->numel(),
            d_x->data<T>());
      }
    } else {
      if (d_scale && d_bias) {
        KeBNBackwardScaleBias<
            T, threads,
            framework::DataLayout::kNHWC><<<grid, threads, 0, stream>>>(
            dy_d, x_d, saved_mean, saved_inv_var, epsilon, N, C, fsize,
            d_scale->data<T>(), d_bias->data<T>());
      }
      if (d_x) {
        KeBNBackwardData<
            T, framework::DataLayout::kNHWC><<<grid2, block, 0, stream>>>(
            dy_d, x_d, scale->data<T>(), saved_mean, saved_inv_var, stats,
            stats + C, stats + 2 * C, epsilon, C, fsize, x->numel(),
            d_x->data<T>());
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm, ops::SyncBatchNormKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormKernel<plat::CUDADeviceContext, double>);
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm_grad,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, double>);
