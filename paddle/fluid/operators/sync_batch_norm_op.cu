/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/sync_batch_norm_op.cu.h"

namespace paddle {
namespace operators {

template <typename T>
class SyncBatchNormKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const std::string layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout layout = framework::StringToDataLayout(layout_str);
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    const bool trainable_stats = ctx.Attr<bool>("trainable_statistics");
    PADDLE_ENFORCE_EQ(use_global_stats, false,
                      platform::errors::InvalidArgument(
                          "sync_batch_norm doesn't support "
                          "to set use_global_stats True. Please use batch_norm "
                          "in this case."));

    const auto *x = ctx.Input<Tensor>("X");
    auto *y = ctx.Output<Tensor>("Y");

    const auto *est_mean = ctx.Input<Tensor>("Mean");
    const auto *est_var = ctx.Input<Tensor>("Variance");

    // moving mean/variance
    auto *mean_out = ctx.Output<Tensor>("MeanOut");
    auto *variance_out = ctx.Output<Tensor>("VarianceOut");

    auto *saved_mean = ctx.Output<Tensor>("SavedMean");
    auto *saved_inv_variance = ctx.Output<Tensor>("SavedVariance");

    bool test_mode = is_test && (!trainable_stats);
    SyncBatchNormFunctor<platform::CUDADeviceContext, T>(
        ctx, layout, x, y, est_mean, est_var, mean_out, variance_out,
        saved_mean, saved_inv_variance, epsilon, momentum, test_mode,
        use_global_stats);
  }
};

template <typename T>
class SyncBatchNormGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const std::string layout_str = ctx.Attr<std::string>("data_layout");

    const DataLayout layout = framework::StringToDataLayout(layout_str);
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");
    const auto *bias = ctx.Input<Tensor>("Bias");

    // init output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    const auto *saved_mean = ctx.Input<Tensor>("SavedMean");
    const auto *saved_inv_var = ctx.Input<Tensor>("SavedVariance");

    SyncBatchNormGradFunctor<platform::CUDADeviceContext, T>(
        ctx, layout, scale, bias, d_x, d_y, d_scale, d_bias, saved_mean,
        saved_inv_var, epsilon);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm, ops::SyncBatchNormKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm_grad,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
#else
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm, ops::SyncBatchNormKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormKernel<plat::CUDADeviceContext, double>,
    ops::SyncBatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm_grad,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, double>,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);
#endif

// clang-format on
