#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

// clang-format off
#include <algorithm>
#include <cfloat>
#include <cmath>
#include <string>
#include <vector>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/operators/batch_norm_op.h"
#include "paddle/fluid/operators/norm_utils.h"
#include "paddle/fluid/platform/cudnn_helper.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/nccl_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using DataLayout = framework::DataLayout;
template <typename T>
using CudnnDataType = platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T, int BlockDim, framework::DataLayout layout>
__global__ void KeLocalStats(const T *x, int N, int M, int C,
                             BatchNormParamType<T> *mean_var) {
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int k = blockIdx.x; k < C; k += gridDim.x) {
    BatchNormParamType<T> x_sum = 0.;
    BatchNormParamType<T> x2_sum = 0.;
    for (int i = threadIdx.x; i < N * M; i += BlockDim) {
      int id = layout == framework::DataLayout::kNCHW
                   ? (i / M) * C * M + k * M + i % M
                   : i * C + k;
      auto x_in = static_cast<BatchNormParamType<T>>(x[id]);
      x_sum += x_in;
      x2_sum += x_in * x_in;
    }
    __syncthreads();
    auto out = BlockReduce(temp_storage).Reduce(x_sum, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      mean_var[k] = out / (N * M);
    }
    out = BlockReduce(temp_storage).Reduce(x2_sum, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      mean_var[k + C] = out / (N * M);
    }
  }
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    mean_var[2 * C] = static_cast<BatchNormParamType<T>>(1.0);
  }
}

template <typename T>
__global__ void KeSyncAndMovingStats(
    BatchNormParamType<T> *means, BatchNormParamType<T> *variances,
    BatchNormParamType<T> *num_dev, const int C,
    const BatchNormParamType<T> momentum, const double epsilon,
    BatchNormParamType<T> *sv_mean_data, BatchNormParamType<T> *sv_inv_var_data,
    BatchNormParamType<T> *moving_means,
    BatchNormParamType<T> *moving_variances) {
  // sync stats across multi-devices
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < C; i += stride) {
    auto mean = means[i] / (*num_dev);
    auto var = variances[i] / (*num_dev);
    var = var - mean * mean;

    // sync stats
    sv_mean_data[i] = mean;
    sv_inv_var_data[i] = 1.0 / sqrt(var + epsilon);
    variances[i] = var;

    // moving stats
    moving_means[i] = moving_means[i] * momentum + mean * (1. - momentum);
    moving_variances[i] =
        moving_variances[i] * momentum + var * (1. - momentum);
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeNormAffine(const T *x,
                                    const BatchNormParamType<T> *scale,
                                    const BatchNormParamType<T> *bias,
                                    const BatchNormParamType<T> *mean,
                                    const BatchNormParamType<T> *variance,
                                    const double epsilon, const int C,
                                    const int M, const int num, T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? (i / M) % C : i % C;
    auto x_i = static_cast<BatchNormParamType<T>>(x[i]);
    auto y_i =
        (x_i - mean[c]) / sqrt(variance[c] + epsilon) * scale[c] + bias[c];
    y[i] = static_cast<T>(y_i);
  }
}

template <typename DeviceContext, typename T>
class SyncBatchNormKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const float momentum = ctx.Attr<float>("momentum");
    const bool is_test = ctx.Attr<bool>("is_test");
    const std::string layout_str = ctx.Attr<std::string>("data_layout");
    const DataLayout layout = framework::StringToDataLayout(layout_str);
    const bool use_global_stats = ctx.Attr<bool>("use_global_stats");
    PADDLE_ENFORCE(
        !use_global_stats,
        "sync_batch_norm doesn't support to set use_global_stats True. ",
        "Please use batch_norm in this case.");

    const auto *x = ctx.Input<Tensor>("X");
    const auto &x_dims = x->dims();
    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, layout, &N, &C, &H, &W, &D);
    int x_numel = x->numel();

    const T *x_d = x->data<T>();
    const auto *s_d = ctx.Input<Tensor>("Scale")->data<BatchNormParamType<T>>();
    const auto *b_d = ctx.Input<Tensor>("Bias")->data<BatchNormParamType<T>>();

    auto *y = ctx.Output<Tensor>("Y");
    T *y_d = y->mutable_data<T>(ctx.GetPlace());

    const BatchNormParamType<T> *mean_data = nullptr;
    const BatchNormParamType<T> *var_data = nullptr;

    auto &dev_ctx = ctx.cuda_device_context();
    auto stream = dev_ctx.stream();
    auto *comm = dev_ctx.nccl_comm();
    const int block = 512;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();

    paddle::memory::AllocationPtr alloc_ptr{nullptr};

    if (is_test) {
      const auto *est_mean = ctx.Input<Tensor>("Mean");
      const auto *est_var = ctx.Input<Tensor>("Variance");
      mean_data = est_mean->data<BatchNormParamType<T>>();
      var_data = est_var->data<BatchNormParamType<T>>();
    } else {
      // x, x^2, 1, here 1 is used to calc device num
      // device num also can be got from platform::DeviceContextPool
      const int bytes = (C * 2 + 1) * sizeof(BatchNormParamType<T>);
      alloc_ptr = memory::Alloc(dev_ctx, bytes);

      auto *stats = reinterpret_cast<BatchNormParamType<T> *>(alloc_ptr->ptr());
      const int threads = 256;
      int grid = std::min(C, (max_threads + threads - 1) / threads);
      if (layout == framework::DataLayout::kNCHW) {
        KeLocalStats<T, threads, framework::DataLayout::kNCHW>
            <<<grid, threads, 0, stream>>>(x_d, N, H * W * D, C, stats);
      } else {
        KeLocalStats<T, threads, framework::DataLayout::kNHWC>
            <<<grid, threads, 0, stream>>>(x_d, N, H * W * D, C, stats);
      }

      // moving mean/variance
      auto *mean_out = ctx.Output<Tensor>("MeanOut");
      auto *variance_out = ctx.Output<Tensor>("VarianceOut");
      auto *est_mean_data =
          mean_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      auto *est_var_data =
          variance_out->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());

      auto *saved_mean = ctx.Output<Tensor>("SavedMean");
      auto *saved_inv_variance = ctx.Output<Tensor>("SavedVariance");
      auto *sv_mean_data =
          saved_mean->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      auto *sv_inv_var_data =
          saved_inv_variance->mutable_data<BatchNormParamType<T>>(
              ctx.GetPlace());

      Tensor c_g_st;
      auto *c_g_st_d = c_g_st.mutable_data<BatchNormParamType<T>>(
          {2 * C + 1}, platform::CPUPlace());
      auto gplace = boost::get<platform::CUDAPlace>(ctx.GetPlace());
      memory::Copy(platform::CPUPlace(), c_g_st_d, gplace, stats, bytes, 0);

      int dtype = platform::ToNCCLDataType(mean_out->type());
      // In-place operation
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::ncclAllReduce(
          stats, stats, 2 * C + 1, static_cast<ncclDataType_t>(dtype), ncclSum,
          comm, stream));

      // Note, Input('Mean')/Input('Variance') share variable with
      // Output('MeanOut')/Output('VarianceOut')
      KeSyncAndMovingStats<T><<<(C + block - 1) / block, block, 0, stream>>>(
          stats, stats + C, stats + 2 * C, C, momentum, epsilon, sv_mean_data,
          sv_inv_var_data, est_mean_data, est_var_data);

      mean_data = sv_mean_data;
      var_data = stats + C;
    }

    int grid2 = (std::min(x_numel, max_threads) + block - 1) / block;
    if (layout == framework::DataLayout::kNCHW) {
      KeNormAffine<T, framework::DataLayout::kNCHW>
          <<<grid2, block, 0, stream>>>(x_d, s_d, b_d, mean_data, var_data,
                                        epsilon, C, H * W * D, x_numel, y_d);
    } else {
      KeNormAffine<T, framework::DataLayout::kNHWC>
          <<<grid2, block, 0, stream>>>(x_d, s_d, b_d, mean_data, var_data,
                                        epsilon, C, H * W * D, x_numel, y_d);
    }
  }
};

template <typename T, const int BlockDim, framework::DataLayout layout>
__global__ void KeBackwardLocalStats(const T *dy, const T *x,
                                     const BatchNormParamType<T> *means, int N,
                                     int M, int C,
                                     BatchNormParamType<T> *sum_dy_prod) {
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  for (int k = blockIdx.x; k < C; k += gridDim.x) {
    BatchNormParamType<T> sum1 = 0.;
    BatchNormParamType<T> sum2 = 0.;
    auto mean = means[k];
    for (int i = threadIdx.x; i < N * M; i += blockDim.x) {
      int id = layout == framework::DataLayout::kNCHW
                   ? (i / M) * C * M + k * M + i % M
                   : i * C + k;
      auto g = static_cast<BatchNormParamType<T>>(dy[id]);
      sum1 += g;
      auto x_i = static_cast<BatchNormParamType<T>>(x[id]);
      sum2 += g * (x_i - mean);
    }

    __syncthreads();
    auto out = BlockReduce(temp_storage).Reduce(sum1, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      sum_dy_prod[k] = out;
    }
    out = BlockReduce(temp_storage).Reduce(sum2, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      sum_dy_prod[k + C] = out;
    }
  }
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    sum_dy_prod[2 * C] = 1.0;
  }
}

template <typename T, int BlockDim, framework::DataLayout layout>
static __global__ void KeBNBackwardScaleBias(
    const T *dy, const T *x, const BatchNormParamType<T> *mean,
    const BatchNormParamType<T> *inv_variance, const double epsilon,
    const int N, const int C, const int HxW, BatchNormParamType<T> *dscale,
    BatchNormParamType<T> *dbias) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> ds_sum = 0.;
    BatchNormParamType<T> db_sum = 0.;

    auto inv_var_i = inv_variance[i];
    auto mean_i = mean[i];
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int id = layout == framework::DataLayout::kNCHW
                         ? ((j / HxW) * C + i) * HxW + (j % HxW)
                         : j * outer_size + i;
      auto x_i = static_cast<BatchNormParamType<T>>(x[id]);
      auto dy_i = static_cast<BatchNormParamType<T>>(dy[id]);
      ds_sum += dy_i * (x_i - mean_i);
      db_sum += dy_i;
    }
    __syncthreads();
    auto os = BlockReduce(temp_storage).Reduce(ds_sum, hipcub::Sum());
    __syncthreads();
    auto ob = BlockReduce(temp_storage).Reduce(db_sum, hipcub::Sum());
    __syncthreads();
    if (threadIdx.x == 0) {
      dscale[i] = os * inv_var_i;
      dbias[i] = ob;
    }
    __syncthreads();
  }
}

template <typename T, framework::DataLayout layout>
static __global__ void KeBNBackwardData(
    const T *dy, const T *x, const BatchNormParamType<T> *gamma,
    const BatchNormParamType<T> *mean,
    const BatchNormParamType<T> *inv_variance,
    const BatchNormParamType<T> *g_sum_dy,
    const BatchNormParamType<T> *g_sum_dy_prod,
    const BatchNormParamType<T> *num_dev, const double epsilon, const int C,
    const int HxW, const int num, T *dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  auto scale = static_cast<BatchNormParamType<T>>(C) / num;
  auto dev_num = num_dev[0];
  for (int i = gid; i < num; i += stride) {
    const int c = layout == framework::DataLayout::kNCHW ? i / HxW % C : i % C;
    auto inv_var = inv_variance[c];
    auto s_d = gamma[c];
    auto gvar =
        -((g_sum_dy_prod[c] / dev_num) * s_d * inv_var * (inv_var * inv_var));
    auto gmean = -((g_sum_dy[c] / dev_num) * s_d * inv_var);

    auto x_i = static_cast<BatchNormParamType<T>>(x[i]);
    auto dy_i = static_cast<BatchNormParamType<T>>(dy[i]);
    auto dx_i =
        dy_i * s_d * inv_var + gmean * scale + gvar * scale * (x_i - mean[c]);
    dx[i] = static_cast<T>(dx_i);
  }
}

// Deriving the Gradient for the Backward Pass of Batch Normalization
// https://kevinzakka.github.io/2016/09/14/batch_normalization/
template <typename DeviceContext, typename T>
class SyncBatchNormGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    double epsilon = static_cast<double>(ctx.Attr<float>("epsilon"));
    const std::string layout_str = ctx.Attr<std::string>("data_layout");

    const DataLayout layout = framework::StringToDataLayout(layout_str);
    const auto *x = ctx.Input<Tensor>("X");
    const auto *d_y = ctx.Input<Tensor>(framework::GradVarName("Y"));
    const auto *scale = ctx.Input<Tensor>("Scale");

    const auto &x_dims = x->dims();

    PADDLE_ENFORCE(x_dims.size() >= 2 && x_dims.size() <= 5,
                   "The Input dim size should be between 2 and 5");
    int N, C, H, W, D;
    ExtractNCWHD(x_dims, layout, &N, &C, &H, &W, &D);

    // init output
    auto *d_x = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto *d_scale = ctx.Output<Tensor>(framework::GradVarName("Scale"));
    auto *d_bias = ctx.Output<Tensor>(framework::GradVarName("Bias"));

    d_x->mutable_data<T>(ctx.GetPlace());
    if (d_scale && d_bias) {
      d_scale->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
      d_bias->mutable_data<BatchNormParamType<T>>(ctx.GetPlace());
    }
    PADDLE_ENFORCE_EQ(scale->dims().size(), 1UL);
    PADDLE_ENFORCE_EQ(scale->dims()[0], C);

    std::vector<int> dims;
    std::vector<int> strides;
    if (layout == DataLayout::kNCHW) {
      dims = {N, C, H, W, D};
      strides = {C * H * W * D, H * W * D, W * D, D, 1};
    } else {
      dims = {N, C, H, W, D};
      strides = {H * W * C * D, 1, W * D * C, D * C, C};
    }

    const T *x_d = x->data<T>();
    const T *dy_d = d_y->data<T>();

    auto &dev_ctx = ctx.cuda_device_context();
    auto stream = dev_ctx.stream();
    auto *comm = dev_ctx.nccl_comm();

    const auto *saved_mean =
        ctx.Input<Tensor>("SavedMean")->data<BatchNormParamType<T>>();
    const auto *saved_inv_var =
        ctx.Input<Tensor>("SavedVariance")->data<BatchNormParamType<T>>();
    const int bytes = (C * 2 + 1) * sizeof(BatchNormParamType<T>);
    auto alloc_ptr = memory::Alloc(dev_ctx, bytes);
    auto *stats = reinterpret_cast<BatchNormParamType<T> *>(alloc_ptr->ptr());

    const int threads = 256;
    int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
    int grid = std::min(C, (max_threads + threads - 1) / threads);
    int x_numel = x->numel();
    int fsize = H * W * D;

    if (layout == framework::DataLayout::kNCHW) {
      KeBackwardLocalStats<T, threads, framework::DataLayout::kNCHW>
          <<<grid, threads, 0, stream>>>(dy_d, x_d, saved_mean, N, fsize, C,
                                         stats);
    } else {
      KeBackwardLocalStats<T, threads, framework::DataLayout::kNHWC>
          <<<grid, threads, 0, stream>>>(dy_d, x_d, saved_mean, N, fsize, C,
                                         stats);
    }
    int dtype = platform::ToNCCLDataType(scale->type());
    // In-place operation
    PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::ncclAllReduce(
        stats, stats, 2 * C + 1, static_cast<ncclDataType_t>(dtype), ncclSum,
        comm, stream));

    const int block = 512;
    int grid2 = (std::min(x_numel, max_threads) + block - 1) / block;
    if (layout == framework::DataLayout::kNCHW) {
      if (d_scale && d_bias) {
        KeBNBackwardScaleBias<T, threads, framework::DataLayout::kNCHW>
            <<<grid, threads, 0, stream>>>(
                dy_d, x_d, saved_mean, saved_inv_var, epsilon, N, C, fsize,
                d_scale->data<BatchNormParamType<T>>(),
                d_bias->data<BatchNormParamType<T>>());
      }
      if (d_x) {
        KeBNBackwardData<T, framework::DataLayout::kNCHW>
            <<<grid2, block, 0, stream>>>(
                dy_d, x_d, scale->data<BatchNormParamType<T>>(), saved_mean,
                saved_inv_var, stats, stats + C, stats + 2 * C, epsilon, C,
                fsize, x->numel(), d_x->data<T>());
      }
    } else {
      if (d_scale && d_bias) {
        KeBNBackwardScaleBias<T, threads, framework::DataLayout::kNHWC>
            <<<grid, threads, 0, stream>>>(
                dy_d, x_d, saved_mean, saved_inv_var, epsilon, N, C, fsize,
                d_scale->data<BatchNormParamType<T>>(),
                d_bias->data<BatchNormParamType<T>>());
      }
      if (d_x) {
        KeBNBackwardData<T, framework::DataLayout::kNHWC>
            <<<grid2, block, 0, stream>>>(
                dy_d, x_d, scale->data<BatchNormParamType<T>>(), saved_mean,
                saved_inv_var, stats, stats + C, stats + 2 * C, epsilon, C,
                fsize, x->numel(), d_x->data<T>());
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm, ops::SyncBatchNormKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormKernel<plat::CUDADeviceContext, double>,
    ops::SyncBatchNormKernel<plat::CUDADeviceContext, plat::float16>);
REGISTER_OP_CUDA_KERNEL(
    sync_batch_norm_grad,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, float>,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, double>,
    ops::SyncBatchNormGradKernel<plat::CUDADeviceContext, plat::float16>);

// clang-format on
