#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/dequantize_log_op.h"
#include "paddle/fluid/operators/math.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/hostdevice.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void KeDequantize(const T* in, const float* dict, int num,
                             float* out) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < num) {
    if (in[idx] < 0) {
      out[idx] = -std::pow(static_cast<float>(2.0), dict[in[idx] + 128]);
    } else {
      out[idx] = std::pow(static_cast<float>(2.0), dict[in[idx]]);
    }
  }
}

template <typename T>
struct DequantizeFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& dev_ctx,
                  const framework::Tensor* in, const framework::Tensor* dict,
                  framework::Tensor* out) {
    const T* in_data = in->data<T>();
    const float* dict_data = dict->data<float>();
    float* out_data = out->mutable_data<float>(dev_ctx.GetPlace());

    int num = in->numel();
    int block = 512;
    int grid = (num + block - 1) / block;

    KeDequantize<T><<<grid, block, 0, dev_ctx.stream()>>>(in_data, dict_data,
                                                          num, out_data);
  }
};

template struct DequantizeFunctor<platform::CUDADeviceContext, int8_t>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
using CUDA = paddle::platform::CUDADeviceContext;
REGISTER_OP_CUDA_KERNEL(dequantize_log, ops::DequantizeLogKernel<CUDA, int8_t>);
