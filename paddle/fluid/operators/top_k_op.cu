#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/top_k_op.h"
#include "paddle/fluid/platform/assert.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
struct Pair {
  __device__ __forceinline__ Pair() {}
  __device__ __forceinline__ Pair(T value, int64_t id) : v(value), id(id) {}

  __device__ __forceinline__ void set(T value, int64_t id) {
    v = value;
    id = id;
  }

  __device__ __forceinline__ void operator=(const Pair<T>& in) {
    v = in.v;
    id = in.id;
  }

  __device__ __forceinline__ bool operator<(const T value) const {
    return (v < value);
  }

  __device__ __forceinline__ bool operator<(const Pair<T>& in) const {
    return (v < in.v) || ((v == in.v) && (id > in.id));
  }

  __device__ __forceinline__ bool operator>(const Pair<T>& in) const {
    return (v > in.v) || ((v == in.v) && (id < in.id));
  }

  T v;
  int64_t id;
};

template <typename T>
__device__ __forceinline__ void AddTo(Pair<T> topk[], const Pair<T>& p,
                                      int beam_size) {
  for (int k = beam_size - 2; k >= 0; k--) {
    if (topk[k] < p) {
      topk[k + 1] = topk[k];
    } else {
      topk[k + 1] = p;
      return;
    }
  }
  topk[0] = p;
}

template <typename T, int beam_size>
__device__ __forceinline__ void AddTo(Pair<T> topk[], const Pair<T>& p) {
  for (int k = beam_size - 2; k >= 0; k--) {
    if (topk[k] < p) {
      topk[k + 1] = topk[k];
    } else {
      topk[k + 1] = p;
      return;
    }
  }
  topk[0] = p;
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* src, int idx,
                                        int dim, int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      AddTo<T>(topk, tmp, beam_size);
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* src, int idx,
                                        int dim, const Pair<T>& max,
                                        int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      if (tmp < max) {
        AddTo<T>(topk, tmp, beam_size);
      }
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* val, int* col,
                                        int idx, int dim, int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < val[idx]) {
      Pair<T> tmp(val[idx], col[idx]);
      AddTo<T>(topk, tmp, beam_size);
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* val, int* col,
                                        int idx, int dim, const Pair<T>& max,
                                        int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < val[idx]) {
      Pair<T> tmp(val[idx], col[idx]);
      if (tmp < max) {
        AddTo<T>(topk, tmp, beam_size);
      }
    }
    idx += BlockSize;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void ThreadGetTopK(Pair<T> topk[], int* beam,
                                              int beam_size, const T* src,
                                              bool* firstStep, bool* is_empty,
                                              Pair<T>* max, int dim,
                                              const int tid) {
  if (*beam > 0) {
    int length = (*beam) < beam_size ? *beam : beam_size;
    if (*firstStep) {
      *firstStep = false;
      GetTopK<T, BlockSize>(topk, src, tid, dim, length);
    } else {
      for (int k = 0; k < MaxLength; k++) {
        if (k < MaxLength - (*beam)) {
          topk[k] = topk[k + *beam];
        } else {
          topk[k].set(-static_cast<T>(INFINITY), -1);
        }
      }
      if (!(*is_empty)) {
        GetTopK<T, BlockSize>(topk + MaxLength - *beam, src, tid, dim, *max,
                              length);
      }
    }

    *max = topk[MaxLength - 1];
    if ((*max).v == -static_cast<T>(1)) *is_empty = true;
    *beam = 0;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void ThreadGetTopK(Pair<T> topk[], int* beam,
                                              int beam_size, const T* val,
                                              int* col, bool* firstStep,
                                              bool* is_empty, Pair<T>* max,
                                              int dim, const int tid) {
  if (*beam > 0) {
    int length = (*beam) < beam_size ? *beam : beam_size;
    if (*firstStep) {
      *firstStep = false;
      GetTopK<T, BlockSize>(topk, val, col, tid, dim, length);
    } else {
      for (int k = 0; k < MaxLength; k++) {
        if (k < MaxLength - *beam) {
          topk[k] = topk[k + *beam];
        } else {
          topk[k].set(-static_cast<T>(INFINITY), -1);
        }
      }
      if (!(*is_empty)) {
        GetTopK<T, BlockSize>(topk + MaxLength - *beam, val, col, tid, dim, max,
                              length);
      }
    }

    *max = topk[MaxLength - 1];
    if ((*max).v == -1) *is_empty = true;
    *beam = 0;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void BlockReduce(Pair<T>* sh_topk, int* maxid,
                                            Pair<T> topk[], T** topVal,
                                            int64_t** topIds, int* beam, int* k,
                                            const int tid, const int warp) {
  while (true) {
    __syncthreads();
    if (tid < BlockSize / 2) {
      if (sh_topk[tid] < sh_topk[tid + BlockSize / 2]) {
        maxid[tid] = tid + BlockSize / 2;
      } else {
        maxid[tid] = tid;
      }
    }
    __syncthreads();
    for (int stride = BlockSize / 4; stride > 0; stride = stride / 2) {
      if (tid < stride) {
        if (sh_topk[maxid[tid]] < sh_topk[maxid[tid + stride]]) {
          maxid[tid] = maxid[tid + stride];
        }
      }
      __syncthreads();
    }
    __syncthreads();

    if (tid == 0) {
      **topVal = sh_topk[maxid[0]].v;
      **topIds = sh_topk[maxid[0]].id;
      (*topVal)++;
      (*topIds)++;
    }
    if (tid == maxid[0]) (*beam)++;
    if (--(*k) == 0) break;
    __syncthreads();

    if (tid == maxid[0]) {
      if (*beam < MaxLength) {
        sh_topk[tid] = topk[*beam];
      }
    }
    // NOTE(zcd): temporary solution
    unsigned mask = 0u;
    CREATE_SHFL_MASK(mask, true);

    if (maxid[0] / 32 == warp) {
      if (platform::CudaShuffleSync(mask, *beam, (maxid[0]) % 32, 32) ==
          MaxLength)
        break;
    }
  }
}

/**
 * Each block compute one sample.
 * In a block:
 * 1. every thread get top MaxLength value;
 * 2. merge to sh_topk, block reduce and get max value;
 * 3. go to the second setp, until one thread's topk value is null;
 * 4. go to the first setp, until get the topk value.
 */

template <typename T, int MaxLength, int BlockSize>
__global__ void KeMatrixTopK(T* output, int output_stride, int64_t* indices,
                             const T* src, int lds, int dim, int k,
                             int grid_dim, int num) {
  __shared__ Pair<T> sh_topk[BlockSize];
  const int tid = threadIdx.x;
  const int warp = threadIdx.x / 32;

  const int bid = blockIdx.x;
  for (int i = bid; i < num; i += grid_dim) {
    int top_num = k;
    __shared__ int maxid[BlockSize / 2];
    T* out = output + i * output_stride;
    int64_t* inds = indices + i * k;
    Pair<T> topk[MaxLength];
    int beam = MaxLength;
    Pair<T> max;
    bool is_empty = false;
    bool firststep = true;

    for (int j = 0; j < MaxLength; j++) {
      topk[j].set(-static_cast<T>(INFINITY), -1);
    }
    while (top_num) {
      ThreadGetTopK<T, MaxLength, BlockSize>(
          topk, &beam, k, src + i * lds, &firststep, &is_empty, &max, dim, tid);

      sh_topk[tid] = topk[0];
      BlockReduce<T, MaxLength, BlockSize>(sh_topk, maxid, topk, &out, &inds,
                                           &beam, &top_num, tid, warp);
    }
  }
}

inline static int GetDesiredBlockDim(int dim) {
  if (dim > 128) {
    return 256;
  } else if (dim > 64) {
    return 128;
  } else if (dim > 32) {
    return 64;
  } else {
    return 32;
  }
}

#define FIXED_BLOCK_DIM_BASE(dim, ...) \
  case (dim): {                        \
    constexpr auto kBlockDim = (dim);  \
    __VA_ARGS__;                       \
  } break

#define FIXED_BLOCK_DIM(...)                \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(32, ##__VA_ARGS__)

template <typename T>
class TopkOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");
    auto* indices = ctx.Output<Tensor>("Indices");
    size_t k = static_cast<int>(ctx.Attr<int>("k"));

    auto* k_t = ctx.Input<Tensor>("K");
    if (k_t) {
      Tensor k_host;
      framework::TensorCopySync(*k_t, platform::CPUPlace(), &k_host);
      k = k_host.data<int>()[0];
      framework::DDim output_dims = output->dims();
      output_dims[output_dims.size() - 1] = k;
      output->Resize(output_dims);
      indices->Resize(output_dims);
    }

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    // FIXME(typhoonzero): data is always converted to type T?
    int64_t* indices_data = indices->mutable_data<int64_t>(ctx.GetPlace());

    framework::DDim inputdims = input->dims();
    const size_t input_height = framework::product(
        framework::slice_ddim(inputdims, 0, inputdims.size() - 1));
    const size_t input_width = inputdims[inputdims.size() - 1];

    if (k > input_width) k = input_width;

    // NOTE: pass lds and dim same to input width.
    // NOTE: old matrix implementation of stride is different to eigen.
    // TODO(typhoonzero): refine this kernel.
    const int kMaxHeight = 2048;
    int gridx = input_height < kMaxHeight ? input_height : kMaxHeight;
    auto& dev_ctx = ctx.cuda_device_context();
    switch (GetDesiredBlockDim(input_width)) {
      FIXED_BLOCK_DIM(
          KeMatrixTopK<T, 5,
                       kBlockDim><<<gridx, kBlockDim, 0, dev_ctx.stream()>>>(
              output_data, k, indices_data, input_data, input_width,
              input_width, static_cast<int>(k), gridx, input_height));
      default:
        PADDLE_THROW("Error");
    }
  }
};

#undef FIXED_BLOCK_DIM_BASE
#undef FIXED_BLOCK_DIM

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    top_k, paddle::operators::TopkOpCUDAKernel<float>,
    paddle::operators::TopkOpCUDAKernel<double>,
    paddle::operators::TopkOpCUDAKernel<paddle::platform::float16>);
