#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <cstdio>
#include "hipcub/hipcub.hpp"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/top_k_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/float16.h"
// set cub base traits in order to handle float16
namespace cub {
template <>
struct NumericTraits<paddle::platform::float16>
    : BaseTraits<FLOATING_POINT, true, false, uint16_t,
                 paddle::platform::float16> {};
}  // namespace cub

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
struct Pair {
  __device__ __forceinline__ Pair() {}
  __device__ __forceinline__ Pair(T value, int64_t id) : v(value), id(id) {}

  __device__ __forceinline__ void set(T value, int64_t id) {
    v = value;
    id = id;
  }

  __device__ __forceinline__ void operator=(const Pair<T>& in) {
    v = in.v;
    id = in.id;
  }

  __device__ __forceinline__ bool operator<(const T value) const {
    return (v < value);
  }

  __device__ __forceinline__ bool operator<(const Pair<T>& in) const {
    return (v < in.v) || ((v == in.v) && (id > in.id));
  }

  __device__ __forceinline__ bool operator>(const Pair<T>& in) const {
    return (v > in.v) || ((v == in.v) && (id < in.id));
  }

  T v;
  int64_t id;
};

template <typename T>
__device__ __forceinline__ void AddTo(Pair<T> topk[], const Pair<T>& p,
                                      int beam_size) {
  for (int k = beam_size - 2; k >= 0; k--) {
    if (topk[k] < p) {
      topk[k + 1] = topk[k];
    } else {
      topk[k + 1] = p;
      return;
    }
  }
  topk[0] = p;
}

template <typename T, int beam_size>
__device__ __forceinline__ void AddTo(Pair<T> topk[], const Pair<T>& p) {
  for (int k = beam_size - 2; k >= 0; k--) {
    if (topk[k] < p) {
      topk[k + 1] = topk[k];
    } else {
      topk[k + 1] = p;
      return;
    }
  }
  topk[0] = p;
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* src, int idx,
                                        int dim, int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      AddTo<T>(topk, tmp, beam_size);
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* src, int idx,
                                        int dim, const Pair<T>& max,
                                        int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      if (tmp < max) {
        AddTo<T>(topk, tmp, beam_size);
      }
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* val, int* col,
                                        int idx, int dim, int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < val[idx]) {
      Pair<T> tmp(val[idx], col[idx]);
      AddTo<T>(topk, tmp, beam_size);
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[], const T* val, int* col,
                                        int idx, int dim, const Pair<T>& max,
                                        int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < val[idx]) {
      Pair<T> tmp(val[idx], col[idx]);
      if (tmp < max) {
        AddTo<T>(topk, tmp, beam_size);
      }
    }
    idx += BlockSize;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void ThreadGetTopK(Pair<T> topk[], int* beam,
                                              int beam_size, const T* src,
                                              bool* firstStep, bool* is_empty,
                                              Pair<T>* max, int dim,
                                              const int tid) {
  if (*beam > 0) {
    int length = (*beam) < beam_size ? *beam : beam_size;
    if (*firstStep) {
      *firstStep = false;
      GetTopK<T, BlockSize>(topk, src, tid, dim, length);
    } else {
      for (int k = 0; k < MaxLength; k++) {
        if (k < MaxLength - (*beam)) {
          topk[k] = topk[k + *beam];
        } else {
          topk[k].set(-static_cast<T>(INFINITY), -1);
        }
      }
      if (!(*is_empty)) {
        GetTopK<T, BlockSize>(topk + MaxLength - *beam, src, tid, dim, *max,
                              length);
      }
    }

    *max = topk[MaxLength - 1];
    if ((*max).v == -static_cast<T>(1)) *is_empty = true;
    *beam = 0;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void ThreadGetTopK(Pair<T> topk[], int* beam,
                                              int beam_size, const T* val,
                                              int* col, bool* firstStep,
                                              bool* is_empty, Pair<T>* max,
                                              int dim, const int tid) {
  if (*beam > 0) {
    int length = (*beam) < beam_size ? *beam : beam_size;
    if (*firstStep) {
      *firstStep = false;
      GetTopK<T, BlockSize>(topk, val, col, tid, dim, length);
    } else {
      for (int k = 0; k < MaxLength; k++) {
        if (k < MaxLength - *beam) {
          topk[k] = topk[k + *beam];
        } else {
          topk[k].set(-static_cast<T>(INFINITY), -1);
        }
      }
      if (!(*is_empty)) {
        GetTopK<T, BlockSize>(topk + MaxLength - *beam, val, col, tid, dim, max,
                              length);
      }
    }

    *max = topk[MaxLength - 1];
    if ((*max).v == -1) *is_empty = true;
    *beam = 0;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void BlockReduce(Pair<T>* sh_topk, int* maxid,
                                            Pair<T> topk[], T** topVal,
                                            int64_t** topIds, int* beam, int* k,
                                            const int tid, const int warp) {
  while (true) {
    __syncthreads();
    if (tid < BlockSize / 2) {
      if (sh_topk[tid] < sh_topk[tid + BlockSize / 2]) {
        maxid[tid] = tid + BlockSize / 2;
      } else {
        maxid[tid] = tid;
      }
    }
    __syncthreads();
    for (int stride = BlockSize / 4; stride > 0; stride = stride / 2) {
      if (tid < stride) {
        if (sh_topk[maxid[tid]] < sh_topk[maxid[tid + stride]]) {
          maxid[tid] = maxid[tid + stride];
        }
      }
      __syncthreads();
    }
    __syncthreads();

    if (tid == 0) {
      **topVal = sh_topk[maxid[0]].v;
      **topIds = sh_topk[maxid[0]].id;
      (*topVal)++;
      (*topIds)++;
    }
    if (tid == maxid[0]) (*beam)++;
    if (--(*k) == 0) break;
    __syncthreads();

    if (tid == maxid[0]) {
      if (*beam < MaxLength) {
        sh_topk[tid] = topk[*beam];
      }
    }
    // NOTE(zcd): temporary solution
    unsigned mask = 0u;
    CREATE_SHFL_MASK(mask, true);

    if (maxid[0] / 32 == warp) {
      if (platform::CudaShuffleSync(mask, *beam, (maxid[0]) % 32, 32) ==
          MaxLength)
        break;
    }
  }
}

/**
 * Each block compute one sample.
 * In a block:
 * 1. every thread get top MaxLength value;
 * 2. merge to sh_topk, block reduce and get max value;
 * 3. go to the second setp, until one thread's topk value is null;
 * 4. go to the first setp, until get the topk value.
 */

template <typename T, int MaxLength, int BlockSize>
__global__ void KeMatrixTopK(T* output, int output_stride, int64_t* indices,
                             const T* src, int lds, int dim, int k,
                             int grid_dim, int num) {
  __shared__ Pair<T> sh_topk[BlockSize];
  const int tid = threadIdx.x;
  const int warp = threadIdx.x / 32;

  const int bid = blockIdx.x;
  for (int i = bid; i < num; i += grid_dim) {
    int top_num = k;
    __shared__ int maxid[BlockSize / 2];
    T* out = output + i * output_stride;
    int64_t* inds = indices + i * k;
    Pair<T> topk[MaxLength];
    int beam = MaxLength;
    Pair<T> max;
    bool is_empty = false;
    bool firststep = true;

    for (int j = 0; j < MaxLength; j++) {
      topk[j].set(-static_cast<T>(INFINITY), -1);
    }
    while (top_num) {
      ThreadGetTopK<T, MaxLength, BlockSize>(
          topk, &beam, k, src + i * lds, &firststep, &is_empty, &max, dim, tid);

      sh_topk[tid] = topk[0];
      BlockReduce<T, MaxLength, BlockSize>(sh_topk, maxid, topk, &out, &inds,
                                           &beam, &top_num, tid, warp);
    }
  }
}

template <typename T, int MaxLength, int BlockSize>
__global__ void AssignGrad(T* x_grad, const int64_t* indices, const T* out_grad,
                           size_t rows, size_t cols, size_t k) {
  for (size_t i = 0; i < rows; ++i) {
    for (size_t j = 0; j < cols; ++j) {
      x_grad[i * cols + j] = 0;
    }
    for (size_t j = 0; j < k; ++j) {
      size_t idx = indices[i * k + j];
      x_grad[i * cols + idx] = out_grad[i * k + j];
    }
  }
}

inline static int GetDesiredBlockDim(int dim) {
  if (dim > 128) {
    return 256;
  } else if (dim > 64) {
    return 128;
  } else if (dim > 32) {
    return 64;
  } else {
    return 32;
  }
}

// Iter for move to next row
struct SegmentOffsetIter {
  EIGEN_DEVICE_FUNC
  explicit SegmentOffsetIter(int num_cols) : num_cols_(num_cols) {}

  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE int operator()(int idx) const {
    return idx * num_cols_;
  }

  int num_cols_;
};

// Iter using into a column
struct ColumnIndexIter {
  explicit ColumnIndexIter(int num_cols) : num_cols_(num_cols) {}

  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE int operator()(
      const Eigen::array<int, 1>& ix) const {
    return ix[0] % num_cols_;
  }

  int num_cols_;
};

__global__ void InitIndex(int64_t* indices, int64_t num_rows,
                          int64_t num_cols) {
  int col_id = threadIdx.x;
  int row_id = blockIdx.x;

  for (int64_t j = row_id; j < num_rows; j += gridDim.x) {
    for (int64_t i = col_id; i < num_cols; i += blockDim.x) {
      indices[j * num_cols + i] = i;
    }
  }
}

template <typename T>
bool SortTopk(const platform::CUDADeviceContext& ctx,
              const framework::Tensor* input_tensor, const int64_t num_cols,
              const int64_t num_rows, const int k,
              framework::Tensor* out_tensor,
              framework::Tensor* indices_tensor) {
  auto cu_stream = ctx.stream();

  Tensor input_indices;
  const std::vector<int64_t> dims = {num_rows, num_cols};
  auto dim = framework::make_ddim(dims);
  input_indices.Resize(dim);
  // input_indices.Resize(num_rows*num_cols);
  input_indices.mutable_data<int64_t>(ctx.GetPlace());
  size_t temp_storage_bytes = -1;

  auto ComputeBlockSize = [](int col) {
    if (col > 512)
      return 1024;
    else if (col > 256 && col <= 512)
      return 512;
    else if (col > 128 && col <= 256)
      return 256;
    else if (col > 64 && col <= 128)
      return 128;
    else
      return 64;
  };

  int block_size = ComputeBlockSize(num_cols);

  unsigned int maxGridDimX = ctx.GetCUDAMaxGridDimSize().x;
  // actually, int num_rows < max_grid_size
  unsigned int grid_size = num_rows < maxGridDimX
                               ? static_cast<unsigned int>(num_rows)
                               : maxGridDimX;
  // Init a index array
  InitIndex<<<grid_size, block_size, 0, cu_stream>>>(
      input_indices.data<int64_t>(), num_rows, num_cols);

  // create iter for counting input
  hipcub::CountingInputIterator<int64_t> counting_iter(0);
  // segment_offset is used for move to next row
  hipcub::TransformInputIterator<int64_t, SegmentOffsetIter,
                              hipcub::CountingInputIterator<int64_t>>
      segment_offsets_t(counting_iter, SegmentOffsetIter(num_cols));

  T* sorted_values_ptr;
  int64_t* sorted_indices_ptr;

  Tensor temp_values;
  Tensor temp_indices;

  const T* input = input_tensor->data<T>();
  T* values = out_tensor->data<T>();
  int64_t* indices = indices_tensor->mutable_data<int64_t>(ctx.GetPlace());

  if (k == num_cols) {
    // Doing a full sort.
    sorted_values_ptr = values;
    sorted_indices_ptr = indices;
  } else {
    temp_values.Resize(dim);
    temp_indices.Resize(dim);
    sorted_values_ptr = temp_values.mutable_data<T>(ctx.GetPlace());
    sorted_indices_ptr = temp_indices.mutable_data<int64_t>(ctx.GetPlace());
  }

  // Get temp storage buffer size, maybe can allocate a fixed buffer to save
  // time.
  auto err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr, temp_storage_bytes, input, sorted_values_ptr,
      input_indices.data<int64_t>(), sorted_indices_ptr, num_cols * num_rows,
      num_rows, segment_offsets_t, segment_offsets_t + 1, 0, sizeof(T) * 8,
      cu_stream);
  if (err != hipSuccess) {
    LOG(ERROR)
        << "TopKOP failed as could not launch "
           "hipcub::DeviceSegmentedRadixSort::SortPairsDescending to calculate "
           "temp_storage_bytes, status: "
        << hipGetErrorString(err);
    return false;
  }
  Tensor temp_storage;
  temp_storage.mutable_data<uint8_t>(ctx.GetPlace(), temp_storage_bytes);

  err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      temp_storage.data<uint8_t>(), temp_storage_bytes, input,
      sorted_values_ptr, input_indices.data<int64_t>(), sorted_indices_ptr,
      num_cols * num_rows, num_rows, segment_offsets_t, segment_offsets_t + 1,
      0, sizeof(T) * 8, cu_stream);
  if (err != hipSuccess) {
    LOG(ERROR)
        << "TopKOP failed as could not launch "
           "hipcub::DeviceSegmentedRadixSort::SortPairsDescending to sort input, "
           "temp_storage_bytes: "
        << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
  auto& dev = *ctx.eigen_device();
  if (k < num_cols) {
    // copy sliced data to output.
    const Eigen::DSizes<Eigen::DenseIndex, 2> slice_indices{0, 0};
    const Eigen::DSizes<Eigen::DenseIndex, 2> slice_sizes{num_rows, k};
    auto e_indices = EigenMatrix<int64_t>::From(*indices_tensor, dim);
    auto e_tmp_indices = EigenMatrix<int64_t>::From(temp_indices);

    std::vector<int> odims = {static_cast<int>(num_rows), static_cast<int>(k)};
    auto dim = framework::make_ddim(odims);
    auto e_values = EigenMatrix<T>::From(*out_tensor, dim);
    auto e_tmp_values = EigenMatrix<T>::From(temp_values);

    e_indices.device(dev) = e_tmp_indices.slice(slice_indices, slice_sizes);
    e_values.device(dev) = e_tmp_values.slice(slice_indices, slice_sizes);
  }
  return true;
}

#define FIXED_BLOCK_DIM_BASE(dim, ...) \
  case (dim): {                        \
    constexpr auto kBlockDim = (dim);  \
    __VA_ARGS__;                       \
  } break

#define FIXED_BLOCK_DIM(...)                \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(32, ##__VA_ARGS__)

template <typename DeviceContext, typename T>
class TopkOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto* input = ctx.Input<Tensor>("X");
    auto* output = ctx.Output<Tensor>("Out");
    auto* indices = ctx.Output<Tensor>("Indices");
    int k = static_cast<int>(ctx.Attr<int>("k"));

    auto* k_t = ctx.Input<Tensor>("K");
    if (k_t) {
      Tensor k_host;
      framework::TensorCopySync(*k_t, platform::CPUPlace(), &k_host);
      k = k_host.data<int>()[0];
      framework::DDim output_dims = output->dims();
      output_dims[output_dims.size() - 1] = k;
      output->Resize(output_dims);
      indices->Resize(output_dims);
    }

    const T* input_data = input->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());
    // FIXME(typhoonzero): data is always converted to type T?

    framework::DDim inputdims = input->dims();
    const int64_t input_height = framework::product(
        framework::slice_ddim(inputdims, 0, inputdims.size() - 1));
    const int64_t input_width = inputdims[inputdims.size() - 1];
    const auto& dev_ctx = ctx.cuda_device_context();

    if ((input_width <= 1024 || k >= 128 || k == input_width)) {
      if (SortTopk<T>(dev_ctx, input, input_width, input_height, k, output,
                      indices)) {
        // Successed, return.
        return;
      } else {
        LOG(INFO) << "TopKOP: Some errors happened when use cub sorting, use "
                     "default topk kernel.";
      }
    }
    int64_t* indices_data = indices->mutable_data<int64_t>(ctx.GetPlace());
    if (k > input_width) k = input_width;

    // NOTE: pass lds and dim same to input width.
    // NOTE: old matrix implementation of stride is different to eigen.
    // TODO(typhoonzero): refine this kernel.
    const int kMaxHeight = 2048;
    int gridx = input_height < kMaxHeight ? input_height : kMaxHeight;
    switch (GetDesiredBlockDim(input_width)) {
      FIXED_BLOCK_DIM(
          KeMatrixTopK<T, 5,
                       kBlockDim><<<gridx, kBlockDim, 0, dev_ctx.stream()>>>(
              output_data, k, indices_data, input_data, input_width,
              input_width, static_cast<int>(k), gridx, input_height));
      default:
        PADDLE_THROW("Error");
    }
  }
};

template <typename DeviceContext, typename T>
class TopkOpGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(context.GetPlace()), true,
        platform::errors::InvalidArgument("It must use CUDAPlace."));
    auto* x = context.Input<Tensor>("X");
    auto* out_grad = context.Input<Tensor>(framework::GradVarName("Out"));
    auto* indices = context.Input<Tensor>("Indices");
    auto* x_grad = context.Output<Tensor>(framework::GradVarName("X"));

    T* x_grad_data = x_grad->mutable_data<T>(context.GetPlace());
    const T* out_grad_data = out_grad->data<T>();
    const int64_t* indices_data = indices->data<int64_t>();
    size_t k = indices->dims()[indices->dims().size() - 1];

    framework::DDim xdims = x->dims();
    const size_t row =
        framework::product(framework::slice_ddim(xdims, 0, xdims.size() - 1));
    const size_t col = xdims[xdims.size() - 1];
    const auto& dev_ctx = context.cuda_device_context();

    const int kMaxHeight = 2048;
    int gridx = row < kMaxHeight ? row : kMaxHeight;
    switch (GetDesiredBlockDim(col)) {
      FIXED_BLOCK_DIM(
          AssignGrad<T, 5,
                     kBlockDim><<<gridx, kBlockDim, 0, dev_ctx.stream()>>>(
              x_grad_data, indices_data, out_grad_data, row, col, k));
      default:
        PADDLE_THROW(
            platform::errors::Unavailable("Error occurs when Assign Grad."));
    }
  }
};
#undef FIXED_BLOCK_DIM_BASE
#undef FIXED_BLOCK_DIM

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(
    top_k,
    paddle::operators::TopkOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        float>,
    paddle::operators::TopkOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        double>,
    paddle::operators::TopkOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        int>,
    paddle::operators::TopkOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        int64_t>,
    paddle::operators::TopkOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                        paddle::platform::float16>);

REGISTER_OP_CUDA_KERNEL(
    top_k_grad,
    paddle::operators::TopkOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                            float>,
    paddle::operators::TopkOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                            double>,
    paddle::operators::TopkOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                            int>,
    paddle::operators::TopkOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                            int64_t>,
    paddle::operators::TopkOpGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                            paddle::platform::float16>);
