#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cmath>
#include "paddle/fluid/memory/buffer.h"
#include "paddle/fluid/operators/optimizers/cast_with_ptr.h"
#include "paddle/fluid/operators/optimizers/distributed_fused_lamb_op.h"
#include "paddle/fluid/operators/tensor_to_string.h"
#include "paddle/fluid/platform/aligned_vector.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/fluid/string/string_helper.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#include "math.h"  // NOLINT
#endif

#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
#include "math.h"  // NOLINT
namespace cub = hipcub;
#endif

namespace paddle {
namespace operators {

template <typename T>
using MasterT = typename details::MPTypeTrait<T>::Type;

template <int LogLevel>
static void LogParamAndTrustRatioDivSquareNorm(
    const framework::ExecutionContext &ctx, const float *param_square_norm,
    const float *trust_ratio_div_square_norm) {
  if (!VLOG_IS_ON(LogLevel)) return;

  auto tensors = ctx.MultiInput<framework::Tensor>("Param");
  if (tensors.empty()) return;

  size_t n = tensors.size();
  auto place = tensors[0]->place();

  auto pn_vec = ToVector(param_square_norm, n, place);
  auto tn_vec = ToVector(trust_ratio_div_square_norm, n, place);

  std::vector<size_t> fp32_indices, fp16_indices;
  fp32_indices.reserve(n);
  fp16_indices.reserve(n);
  for (size_t i = 0; i < n; ++i) {
    const auto *t = tensors[i];
    if (t->type() == framework::proto::VarType::FP32) {
      fp32_indices.push_back(i);
    } else if (t->type() == framework::proto::VarType::FP16) {
      fp16_indices.push_back(i);
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupported data type %s.", framework::DataTypeToString(t->type())));
    }
  }

  for (auto idx : fp16_indices) {
    fp32_indices.push_back(idx);
  }

  const auto &names = ctx.GetOp().Inputs("Param");
  for (size_t i = 0; i < fp32_indices.size(); ++i) {
    auto idx = fp32_indices[i];
    auto dtype = framework::DataTypeToString(tensors[idx]->type());
    VLOG(LogLevel) << "Param " << dtype << " " << names[idx]
                   << " pn = " << pn_vec[i] << " , tn = " << tn_vec[i];
  }
}

static bool IsFinite(const platform::CUDADeviceContext &dev_ctx,
                     const float *ptr) {
  auto stream = dev_ctx.stream();
  float cpu_value;
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&cpu_value, ptr, sizeof(float),
                                            hipMemcpyDeviceToHost, stream));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&cpu_value, ptr, sizeof(float),
                                             hipMemcpyDeviceToHost, stream));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#endif
  LOG(INFO) << "NAN_INF indicator value: " << cpu_value;
  return isfinite(cpu_value);
}

template <typename T>
static const T *GetInputTensorPtr(const framework::ExecutionContext &ctx,
                                  const char *in_name,
                                  int64_t *numel = nullptr) {
  const auto *in_tensor = ctx.Input<framework::Tensor>(in_name);
  PADDLE_ENFORCE_NOT_NULL(in_tensor, platform::errors::InvalidArgument(
                                         "Input(%s) cannot be NULL.", in_name));
  if (in_tensor->IsInitialized()) {
    if (numel) *numel = in_tensor->numel();
    return in_tensor->data<T>();
  } else {
    if (numel) *numel = 0;
    return nullptr;
  }
}

template <typename T, bool AllowNotExist = false>
static T *GetSameInOutTensorPtr(const framework::ExecutionContext &ctx,
                                const platform::Place &place,
                                const char *in_name, const char *out_name,
                                int64_t *numel = nullptr) {
  const auto *in_tensor = ctx.Input<framework::Tensor>(in_name);
  if (in_tensor == nullptr || !in_tensor->IsInitialized()) {
    PADDLE_ENFORCE_EQ(AllowNotExist, true,
                      platform::errors::InvalidArgument(
                          "Input(%s) cannot be NULL.", in_name));
    if (numel) *numel = 0;
    return nullptr;
  }

  auto *out_tensor = ctx.Output<framework::Tensor>(out_name);
  PADDLE_ENFORCE_NOT_NULL(in_tensor, platform::errors::InvalidArgument(
                                         "Input(%s) cannot be NULL.", in_name));
  PADDLE_ENFORCE_NOT_NULL(out_tensor,
                          platform::errors::InvalidArgument(
                              "Output(%s) cannot be NULL.", out_name));
  const T *in_data = in_tensor->data<T>();
  T *out_data = out_tensor->mutable_data<T>(place);
  PADDLE_ENFORCE_EQ(in_data, out_data,
                    platform::errors::InvalidArgument(
                        "Input(%s) and Output(%s) must be the same Tensor.",
                        in_name, out_name));
  if (numel) *numel = out_tensor->numel();
  return out_data;
}

template <typename T>
struct SquareFunctor {
  HOSTDEVICE MasterT<T> operator()(T x) const {
    auto y = static_cast<MasterT<T>>(x);
    return y * y;
  }
};

template <typename T>
struct IsNanInfFunctor {
  HOSTDEVICE bool operator()(T x) const { return !isfinite(x); }
};

struct OrFunctor {
  HOSTDEVICE bool operator()(bool x, bool y) const { return x || y; }
};

struct AndFunctor {
  HOSTDEVICE bool operator()(bool x, bool y) const { return x && y; }
};

template <typename T1, typename T2>
static __global__ void ScaleCUDAKernel(const T1 *__restrict__ x,
                                       const T2 *__restrict__ scale,
                                       T1 *__restrict__ y, int num) {
  static_assert(sizeof(T1) <= sizeof(T2),
                "sizeof(T1) must be not greater than sizeof(T2).");
  T2 s = scale[0];
  CUDA_KERNEL_LOOP(i, num) {
    y[i] = static_cast<T1>(static_cast<T2>(x[i]) * s);
  }
}

template <typename T>
static __global__ void AddToCUDAKernel(const T *__restrict__ x,
                                       T *__restrict__ y) {
  y[0] += x[0];
}

// If clip before allreduce,
// coeff = global_scale * max_global_grad_norm / (1e-6 + sqrt(square_grad_norm)
// * rescale_grad)
// if coeff >= 1 or coeff is Nan/Inf, scale = 1.0
// else scale = coeff
template <typename T1, typename T2>
static __global__ void CalcGradNormClipBeforeAllReduceScale(
    const T1 *__restrict__ global_scale, T1 max_global_grad_norm,
    const T1 *__restrict__ square_grad_norm, T1 *__restrict__ out1,
    T2 *__restrict__ out2, T1 clip_rescale_grad) {
  T1 grad_norm = static_cast<T1>(sqrt(*square_grad_norm)) * clip_rescale_grad;
  T1 scale = global_scale[0] * max_global_grad_norm / (1e-6 + grad_norm);
  bool found_nan_inf = !isfinite(scale);
  if (scale >= 1 || found_nan_inf) {
    scale = static_cast<T1>(1.0);
  }

  if (out1) {
    *out1 = scale;
  }
  if (out2) {
    *out2 = static_cast<T2>(scale);
  }
}

static __global__ void SetNanInfValueCUDAKernelOneFlag(const bool *in_flag_p,
                                                       float *out_p) {
  *out_p = (*in_flag_p) ? __int_as_float(0x7fffffffU) : 0.0f;
}

static __global__ void SetNanInfValueCUDAKernelTwoFlag(const bool *in_flag_p_1,
                                                       const bool *in_flag_p_2,
                                                       float *out_p) {
  *out_p =
      ((*in_flag_p_1) || (*in_flag_p_2)) ? __int_as_float(0x7fffffffU) : 0.0f;
}

// TODO(zengjinle): Vectorize this function
// NOTE: this method does not update Beta1Pow and Beta2Pow!
template <typename T, typename GradT, typename IndexT>
static __global__ void UpdateLambMoment(
    const T *__restrict__ param_p, const GradT *__restrict__ grad_p,
    const T *__restrict__ square_grad_norm_p,
    const T *__restrict__ global_scale, const IndexT *__restrict__ indices,
    const T *__restrict__ weight_decay_p, const T *__restrict__ beta1pow_p,
    const T *__restrict__ beta2pow_p, T *__restrict__ mom1_p,
    T *__restrict__ mom2_p, T *__restrict__ trust_ratio_div_p, T beta1, T beta2,
    T epsilon, T max_global_grad_norm, int num, T rescale_grad) {
  T square_grad_norm = *square_grad_norm_p;
  if (!isfinite(square_grad_norm)) return;

  T scale = rescale_grad / global_scale[0];
  if (max_global_grad_norm > 0) {
    T clip_scale =
        max_global_grad_norm / (sqrtf(square_grad_norm) * scale + 1e-6);
    if (clip_scale < static_cast<T>(1)) {
      scale *= clip_scale;
    }
  }

  T one_minus_beta1pow = 1 - beta1pow_p[0];
  T one_minus_beta2pow = 1 - beta2pow_p[0];

  CUDA_KERNEL_LOOP(i, num) {
    T p = param_p[i];
    T g = static_cast<T>(grad_p[i]) * scale;
    T weight_decay = weight_decay_p[i];
    T mom1 = mom1_p[i];
    T mom2 = mom2_p[i];

    mom1 = beta1 * mom1 + (1 - beta1) * g;
    mom2 = beta2 * mom2 + (1 - beta2) * g * g;

    T mom1_unbiased = mom1 / one_minus_beta1pow;
    T mom2_unbiased = mom2 / one_minus_beta2pow;
    T trust_ratio_div =
        mom1_unbiased / (sqrtf(mom2_unbiased) + epsilon) + weight_decay * p;

    mom1_p[i] = mom1;
    mom2_p[i] = mom2;
    trust_ratio_div_p[i] = trust_ratio_div;
  }
}

template <typename T, bool NeedUpdate /*=true*/>
struct LambBetaPowUpdateOnceHelper {
  LambBetaPowUpdateOnceHelper(T *beta1pow, T *beta2pow, T beta1, T beta2) {
    PADDLE_ENFORCE_NOT_NULL(beta1pow,
                            platform::errors::InvalidArgument(
                                "The beta1pow should not be nullptr."));
    PADDLE_ENFORCE_NOT_NULL(beta2pow,
                            platform::errors::InvalidArgument(
                                "The beta2pow should not be nullptr."));
    beta1pow_ = beta1pow;
    beta2pow_ = beta2pow;
    beta1_ = beta1;
    beta2_ = beta2;
  }

  HOSTDEVICE void UpdateBetaPows() const {
    beta1pow_[0] *= beta1_;
    beta2pow_[0] *= beta2_;
  }

 private:
  T *__restrict__ beta1pow_;
  T *__restrict__ beta2pow_;
  T beta1_;
  T beta2_;
};

template <typename T>
struct LambBetaPowUpdateOnceHelper<T, false> {
  LambBetaPowUpdateOnceHelper(T *beta1pow, T *beta2pow, T beta1, T beta2) {
    PADDLE_ENFORCE_EQ(
        beta1pow, nullptr,
        platform::errors::InvalidArgument("The beta1pow should be nullptr."));
    PADDLE_ENFORCE_EQ(
        beta2pow, nullptr,
        platform::errors::InvalidArgument("The beta2pow should be nullptr."));
  }

  HOSTDEVICE void UpdateBetaPows() const {}
};

template <bool HasFoundInf /*=true*/>
struct LambFoundInfHelper {
 public:
  explicit LambFoundInfHelper(bool *found_inf) : found_inf_(found_inf) {
    PADDLE_ENFORCE_NOT_NULL(found_inf,
                            platform::errors::InvalidArgument(
                                "The found_inf should not be nullptr."));
  }

  HOSTDEVICE void UpdateFoundInf(bool value) { *found_inf_ = value; }

 private:
  bool *__restrict__ found_inf_;
};

template <>
struct LambFoundInfHelper<false> {
 public:
  explicit LambFoundInfHelper(bool *found_inf) {
    PADDLE_ENFORCE_EQ(
        found_inf, nullptr,
        platform::errors::InvalidArgument("The found_inf should be nullptr."));
  }

  HOSTDEVICE void UpdateFoundInf(bool) {}
};

template <typename T, bool HasMasterParam /*=true*/>
struct LambParamHelper {
  LambParamHelper(T *param, MasterT<T> *master_param) {
    constexpr bool kIsSameType = std::is_same<T, MasterT<T>>::value;
    PADDLE_ENFORCE_EQ(kIsSameType, false,
                      platform::errors::InvalidArgument(
                          "T must not be the same with MasterT<T>."));
    PADDLE_ENFORCE_NOT_NULL(master_param,
                            platform::errors::InvalidArgument(
                                "Master parameter must be provided."));
    param_ = param;
    master_param_ = master_param;
  }

  HOSTDEVICE void SetParam(int i, MasterT<T> updated_p) {
    param_[i] = static_cast<T>(updated_p);
    master_param_[i] = updated_p;
  }

  HOSTDEVICE MasterT<T> GetParam(int i) { return master_param_[i]; }

 private:
  T *__restrict__ param_;
  MasterT<T> *__restrict__ master_param_;
};

template <typename T>
struct LambParamHelper<T, false> {
  LambParamHelper(T *param, MasterT<T> *master_param) {
    constexpr bool kIsSameType = std::is_same<T, MasterT<T>>::value;
    PADDLE_ENFORCE_EQ(kIsSameType, true,
                      platform::errors::InvalidArgument(
                          "T must be the same with MasterT<T>."));
    if (master_param != nullptr) {
      PADDLE_ENFORCE_EQ(static_cast<void *>(param),
                        static_cast<void *>(master_param),
                        platform::errors::InvalidArgument(
                            "Master parameter must be nullptr or the same as "
                            "non-master parameter."));
    }
    param_ = param;
  }

  HOSTDEVICE void SetParam(int i, MasterT<T> updated_p) {
    param_[i] = static_cast<T>(updated_p);
  }

  HOSTDEVICE MasterT<T> GetParam(int i) {
    return static_cast<MasterT<T>>(param_[i]);
  }

 private:
  T *__restrict__ param_;
};

template <typename ParamT, typename IndexT, bool HasMasterParam,
          bool NeedUpdateBetaPow, bool HasFoundInf>
struct LambParamAndBetaPowsUpdateHelper
    : public LambParamHelper<ParamT, HasMasterParam>,
      public LambBetaPowUpdateOnceHelper<MasterT<ParamT>, NeedUpdateBetaPow>,
      public LambFoundInfHelper<HasFoundInf> {
  LambParamAndBetaPowsUpdateHelper(
      ParamT *param, MasterT<ParamT> *master_param, MasterT<ParamT> *beta1pow,
      MasterT<ParamT> *beta2pow, MasterT<ParamT> beta1, MasterT<ParamT> beta2,
      bool *found_inf, const MasterT<ParamT> *trust_ratio_div,
      const MasterT<ParamT> *lr, const IndexT *index,
      const MasterT<ParamT> *param_square_norm,
      const MasterT<ParamT> *trust_ratio_div_square_norm,
      const MasterT<ParamT> *update_flag)
      : LambParamHelper<ParamT, HasMasterParam>(param, master_param),
        LambBetaPowUpdateOnceHelper<MasterT<ParamT>, NeedUpdateBetaPow>(
            beta1pow, beta2pow, beta1, beta2),
        LambFoundInfHelper<HasFoundInf>(found_inf),
        trust_ratio_div(trust_ratio_div),
        lr(lr),
        index(index),
        param_square_norm(param_square_norm),
        trust_ratio_div_square_norm(trust_ratio_div_square_norm),
        update_flag(update_flag) {}

  const MasterT<ParamT> *__restrict__ trust_ratio_div;
  const MasterT<ParamT> *__restrict__ lr;
  const IndexT *__restrict__ index;
  const MasterT<ParamT> *__restrict__ param_square_norm;
  const MasterT<ParamT> *__restrict__ trust_ratio_div_square_norm;
  const MasterT<ParamT> *__restrict__ update_flag;
};

template <typename ParamT, typename IndexT, bool HasMasterParam,
          bool NeedUpdateBetaPow, bool HasFoundInf>
static __global__ void LambUpdateParamAndBetaPowsCUDAKernel(
    LambParamAndBetaPowsUpdateHelper<ParamT, IndexT, HasMasterParam,
                                     NeedUpdateBetaPow, HasFoundInf>
        args,
    int num) {
  auto should_update = *args.update_flag;
  if (!isfinite(should_update)) {
    if (HasFoundInf && threadIdx.x == 0 && blockIdx.x == 0) {
      args.UpdateFoundInf(true);
    }
    return;
  } else if (HasFoundInf && threadIdx.x == 0 && blockIdx.x == 0) {
    args.UpdateFoundInf(false);
  }

  if (NeedUpdateBetaPow && threadIdx.x == 0 && blockIdx.x == 0) {
    args.UpdateBetaPows();
  }

  using MT = MasterT<ParamT>;

  MT lr_value = *args.lr;
  CUDA_KERNEL_LOOP(i, num) {
    MT p = args.GetParam(i);
    MT t = args.trust_ratio_div[i];
    auto norm_idx = args.index[i];
    MT p_square_norm = args.param_square_norm[norm_idx];
    if (p_square_norm < static_cast<MT>(0)) {
      asm("trap;");
    }

    MT t_square_norm = args.trust_ratio_div_square_norm[norm_idx];

    MT p_norm = static_cast<MT>(sqrtf(p_square_norm));
    MT t_norm = static_cast<MT>(sqrtf(t_square_norm));

    auto update = (p_norm != static_cast<MT>(0) && t_norm != static_cast<MT>(0))
                      ? p_norm / t_norm
                      : static_cast<MT>(1);

    MT updated_p = p - lr_value * update * t;
    args.SetParam(i, updated_p);
  }
}

template <typename ParamT, typename IndexT>
static void LambUpdateParamAndBetaPows(
    const platform::CUDADeviceContext &dev_ctx,
    const MasterT<ParamT> *trust_ratio_div, const MasterT<ParamT> *lr,
    const IndexT *index, const MasterT<ParamT> *param_square_norm,
    const MasterT<ParamT> *trust_ratio_div_square_norm,
    const MasterT<ParamT> *update_flag, MasterT<ParamT> **beta1pow,
    MasterT<ParamT> **beta2pow, bool **found_inf, MasterT<ParamT> beta1,
    MasterT<ParamT> beta2, int num, ParamT *param,
    MasterT<ParamT> *master_param, gpuStream_t stream) {
  if (num == 0) return;

  bool has_master_param = !(std::is_same<ParamT, MasterT<ParamT>>::value);
  auto has_beta_pow = (*beta1pow) != nullptr && (*beta2pow) != nullptr;
  auto has_found_inf = (*found_inf) != nullptr;

#define PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(                              \
    __has_master_param, __has_beta_pow, __has_found_inf)                     \
  do {                                                                       \
    LambParamAndBetaPowsUpdateHelper<ParamT, IndexT, __has_master_param,     \
                                     __has_beta_pow, __has_found_inf>        \
        helper(param, master_param, *beta1pow, *beta2pow, beta1, beta2,      \
               *found_inf, trust_ratio_div, lr, index, param_square_norm,    \
               trust_ratio_div_square_norm, update_flag);                    \
    auto config = platform::GetGpuLaunchConfig1D(dev_ctx, num);              \
    LambUpdateParamAndBetaPowsCUDAKernel<<<                                  \
        config.block_per_grid, config.thread_per_block, 0, stream>>>(helper, \
                                                                     num);   \
  } while (0)

  if (has_master_param) {
    if (has_beta_pow) {
      if (has_found_inf) {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(true, true, true);
      } else {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(true, true, false);
      }
    } else {
      if (has_found_inf) {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(true, false, true);
      } else {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(true, false, false);
      }
    }
  } else {
    if (has_beta_pow) {
      if (has_found_inf) {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(false, true, true);
      } else {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(false, true, false);
      }
    } else {
      if (has_found_inf) {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(false, false, true);
      } else {
        PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL(false, false, false);
      }
    }
  }

  *beta1pow = nullptr;
  *beta2pow = nullptr;
  *found_inf = nullptr;
#undef PADDLE_LAUNCH_LAMB_UPDATE_PARAM_KERNEL
}

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
static bool CreatePreMulScaleOpIfSupported(ncclDataType_t dtype,
                                           ncclComm_t comm, const void *scale,
                                           ncclRedOp_t *op) {
#if NCCL_VERSION_CODE >= 21100
  int ver;
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclGetVersion(&ver));
  if (ver >= 21100) {
    VLOG(10) << "ncclRedOpCreatePreMulSum is supported.";
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclRedOpCreatePreMulSum(
        op, const_cast<void *>(scale), dtype, ncclScalarDevice, comm));
    return true;
  }
#endif
  VLOG(10) << "ncclRedOpCreatePreMulSum is not supported.";
  return false;
}

template <typename T>
static void NCCLReduceScatterWithScale(
    const T *sendbuff, T *recvbuff, size_t recvcount, size_t nranks,
    ncclComm_t comm, gpuStream_t stream,
    const platform::CUDADeviceContext &dev_ctx, const T *scale = nullptr) {
  static_assert(std::is_same<T, float>::value ||
                    std::is_same<T, platform::float16>::value,
                "T must be either float32 or float16.");
  if (recvcount == 0) return;

  if (comm == nullptr) {
    if (scale != nullptr) {
      PADDLE_ENFORCE_EQ(nranks, 1,
                        platform::errors::InvalidArgument(
                            "nranks must be 1 when scale != nullptr."));
      auto numel = recvcount * nranks;
      auto config = platform::GetGpuLaunchConfig1D(dev_ctx, numel);
      ScaleCUDAKernel<<<config.block_per_grid, config.thread_per_block, 0,
                        stream>>>(sendbuff, scale, recvbuff, numel);
    }
    return;
  }

  ncclRedOp_t op = ncclSum;
  ncclDataType_t dtype =
      std::is_same<T, float>::value ? ncclFloat32 : ncclFloat16;
  bool should_destroy_op =
      scale && CreatePreMulScaleOpIfSupported(dtype, comm, scale, &op);
  memory::Buffer buffer(dev_ctx.GetPlace());
  if (scale && !should_destroy_op) {
    size_t numel = recvcount * nranks;
    T *new_sendbuff = buffer.Alloc<T>(numel);
    auto config = platform::GetGpuLaunchConfig1D(dev_ctx, numel);
    ScaleCUDAKernel<<<config.block_per_grid, config.thread_per_block, 0,
                      stream>>>(sendbuff, scale, new_sendbuff, numel);
    sendbuff = new_sendbuff;
  }

  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclReduceScatter(
      sendbuff, recvbuff, recvcount, dtype, op, comm, stream));

#if NCCL_VERSION_CODE >= 21100
  if (should_destroy_op) {
    VLOG(10) << "ncclRedOpDestroy starts";
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclRedOpDestroy(op, comm));
    VLOG(10) << "ncclRedOpDestroy ends";
  }
#endif
}
#endif

template <typename InputIteratorT, typename OutputIteratorT, typename ReduceOpT,
          typename T>
static void CubDeviceReduce(InputIteratorT d_in, OutputIteratorT d_out,
                            int num_items, ReduceOpT reduction_op, T init,
                            gpuStream_t stream, memory::Buffer *buffer) {
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                num_items, reduction_op, init, stream));
  d_temp_storage = buffer->Alloc<void>(temp_storage_bytes);
  VLOG(10) << "hipcub::DeviceReduce::Reduce needs " << temp_storage_bytes
           << " byte(s), ptr = " << d_temp_storage;
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                num_items, reduction_op, init, stream));
}

template <typename InputIteratorT, typename OutputIteratorT,
          typename OffsetIteratorT, typename ReductionOp, typename T>
static void CubDeviceSegmentedReduce(InputIteratorT d_in, OutputIteratorT d_out,
                                     int num_segments,
                                     OffsetIteratorT d_begin_offsets,
                                     OffsetIteratorT d_end_offsets,
                                     ReductionOp reduction_op, T initial_value,
                                     gpuStream_t stream,
                                     memory::Buffer *buffer) {
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedReduce::Reduce(
      d_temp_storage, temp_storage_bytes, d_in, d_out, num_segments,
      d_begin_offsets, d_end_offsets, reduction_op, initial_value, stream));
  d_temp_storage = buffer->Alloc<void>(temp_storage_bytes);
  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedReduce::Reduce(
      d_temp_storage, temp_storage_bytes, d_in, d_out, num_segments,
      d_begin_offsets, d_end_offsets, reduction_op, initial_value, stream));
}

template <typename T>
struct AddConstantFunctor {
  explicit AddConstantFunctor(T bias) : bias_(bias) {}

  T operator()(T x) const { return x + bias_; }

 private:
  T bias_;
};

template <typename T>
struct OffsetWithBiasFunctor {
  OffsetWithBiasFunctor(const T *offset, T bias)
      : offset_(offset), bias_(bias) {}

  HOSTDEVICE T operator()(T idx) const { return offset_[idx] - bias_; }

  HOSTDEVICE constexpr bool operator==(const OffsetWithBiasFunctor<T> &) const {
    return true;
  }

 private:
  const T *offset_;
  const T bias_;
};

template <typename T, typename OffsetT>
static void CubDeviceSegmentedSquareNorm(const T *x, MasterT<T> *y, int n,
                                         const OffsetT *offset,
                                         OffsetT init_offset,
                                         gpuStream_t stream,
                                         memory::Buffer *buffer) {
  if (n <= 0) return;
  hipcub::TransformInputIterator<MasterT<T>, SquareFunctor<T>, const T *> iter(
      x, SquareFunctor<T>());
  if (init_offset == static_cast<OffsetT>(0)) {
    CubDeviceSegmentedReduce(iter, y, n, offset, offset + 1, hipcub::Sum(),
                             static_cast<MasterT<T>>(0), stream, buffer);
  } else {
    hipcub::CountingInputIterator<OffsetT> cnt_iter(0);
    OffsetWithBiasFunctor<OffsetT> functor(offset, init_offset);
    hipcub::TransformInputIterator<OffsetT, OffsetWithBiasFunctor<OffsetT>,
                                hipcub::CountingInputIterator<OffsetT>>
        offset_iter(cnt_iter, functor);
    CubDeviceSegmentedReduce(iter, y, n, offset_iter, offset_iter + 1,
                             hipcub::Sum(), static_cast<MasterT<T>>(0), stream,
                             buffer);
  }
}

template <typename T>
static void GetSquareGradNormImpl(const T *grad, int n, float *square_norm,
                                  gpuStream_t stream,
                                  memory::Buffer *cub_tmp_buffer) {
  using Iterator =
      hipcub::TransformInputIterator<float, SquareFunctor<T>, const T *>;
  Iterator iter(grad, SquareFunctor<T>());
  CubDeviceReduce(iter, square_norm, n, hipcub::Sum(), static_cast<float>(0),
                  stream, cub_tmp_buffer);
}

// square_norm is of length 2 at least
static void GetSquareGradNorm(const float *fp32_grad, int fp32_numel,
                              const platform::float16 *fp16_grad,
                              int fp16_numel, float *square_norm,
                              gpuStream_t stream,
                              memory::Buffer *cub_tmp_buffer) {
  VLOG(10) << "GetSquareGradNorm starts, fp32_numel = " << fp32_numel
           << " , fp16_numel = " << fp16_numel;
  if (fp32_numel > 0) {
    GetSquareGradNormImpl(fp32_grad, fp32_numel, square_norm, stream,
                          cub_tmp_buffer);
    VLOG(10) << "FP32 square L2-Norm: "
             << FlattenToString(square_norm, 1, cub_tmp_buffer->GetPlace());
  }

  if (fp16_numel > 0) {
    float *fp16_square_norm = fp32_numel > 0 ? square_norm + 1 : square_norm;
    GetSquareGradNormImpl(fp16_grad, fp16_numel, fp16_square_norm, stream,
                          cub_tmp_buffer);
    VLOG(10) << "FP16 square L2-Norm: "
             << FlattenToString(fp16_square_norm, 1,
                                cub_tmp_buffer->GetPlace());
    if (fp32_numel > 0) {
      AddToCUDAKernel<<<1, 1, 0, stream>>>(fp16_square_norm, square_norm);
      VLOG(10) << "FP32+FP16 square L2-Norm: "
               << FlattenToString(square_norm, 1, cub_tmp_buffer->GetPlace());
    }
  }
  VLOG(10) << "GetSquareGradNorm ends, fp32_numel = " << fp32_numel
           << " , fp16_numel = " << fp16_numel;
}

template <typename T>
std::string NumToString(T x) {
  std::stringstream ss;
  ss << x;
  return ss.str();
}

template <typename T>
static std::string GetMinMaxStr(const T *x, size_t n,
                                const platform::Place &place) {
  PADDLE_ENFORCE_EQ(
      platform::is_gpu_place(place), true,
      platform::errors::InvalidArgument("Only support CUDAPlace currently."));

  auto *dev_ctx = static_cast<platform::CUDADeviceContext *>(
      platform::DeviceContextPool::Instance().Get(place));
  auto stream = dev_ctx->stream();

  memory::Buffer ret_buffer(place);
  T *ret = ret_buffer.Alloc<T>(2);

  if (n > 0) {
    memory::Buffer cub_buffer(place);
    CubDeviceReduce(x, ret, n, hipcub::Min(), std::numeric_limits<T>::max(),
                    stream, &cub_buffer);
    CubDeviceReduce(x, ret + 1, n, hipcub::Max(), std::numeric_limits<T>::lowest(),
                    stream, &cub_buffer);
    T ret_cpu[2];
#ifdef PADDLE_WITH_HIP
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&ret_cpu[0], ret, 2 * sizeof(T),
                                              hipMemcpyDeviceToHost, stream));
    PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&ret_cpu[0], ret, 2 * sizeof(T),
                                               hipMemcpyDeviceToHost, stream));
    PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#endif
    return std::string("{\"min\": ") + NumToString(ret_cpu[0]) +
           " , \"max\": " + NumToString(ret_cpu[1]) + "}";
  } else {
    return "{\"min\": null, \"max\": null}";
  }
}

struct VisitDTypeFunctor {
  VisitDTypeFunctor(const framework::Tensor *x, std::string *s)
      : x_(x), s_(s) {}

  template <typename T>
  void apply() const {
    *s_ = GetMinMaxStr<T>(x_->template data<T>(), x_->numel(), x_->place());
  }

 private:
  const framework::Tensor *x_;
  std::string *s_;
};

static std::string GetMinMaxStr(const framework::Tensor *x) {
  if (x == nullptr) return "null";
  if (!x->IsInitialized()) return "not_inited";
  if (!platform::is_gpu_place(x->place())) return "CPUTensor";
  std::string str;
  VisitDTypeFunctor functor(x, &str);
  framework::VisitDataType(x->type(), functor);
  return str;
}

static void PrintAllMinMaxRange(const framework::ExecutionContext &ctx,
                                bool only_inputs) {
  if (!VLOG_IS_ON(1)) return;
  for (const auto &pair : ctx.GetOp().Inputs()) {
    const auto &key = pair.first;
    const auto tensors = ctx.MultiInput<framework::Tensor>(key);
    size_t n = tensors.size();
    for (size_t i = 0; i < n; ++i) {
      VLOG(1) << "Input(" << key + ")[" << i << "] = " << pair.second[i]
              << " , " << GetMinMaxStr(tensors[i]);
    }
  }

  if (only_inputs) return;
  for (const auto &pair : ctx.GetOp().Outputs()) {
    const auto &key = pair.first;
    const auto tensors = ctx.MultiOutput<framework::Tensor>(key);
    size_t n = tensors.size();
    for (size_t i = 0; i < n; ++i) {
      VLOG(1) << "Output(" << key + ")[" << i << "] = " << pair.second[i]
              << " , " << GetMinMaxStr(tensors[i]);
    }
  }
}

template <typename T>
static bool HasNanInf(const T *x, int n, gpuStream_t stream,
                      memory::Buffer *cub_tmp_buffer) {
  if (n <= 0) return false;
  memory::Buffer buffer(cub_tmp_buffer->GetPlace());
  auto *flag = buffer.Alloc<bool>(1);
  hipcub::TransformInputIterator<bool, IsNanInfFunctor<T>, const T *> iter(
      x, IsNanInfFunctor<T>());
  CubDeviceReduce(iter, flag, n, OrFunctor(), false, stream, cub_tmp_buffer);
  bool cpu_flag = false;
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&cpu_flag, flag, sizeof(bool),
                                             hipMemcpyDeviceToHost, stream));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
  return cpu_flag;
}

static void CheckHasNanInfGrad(const float *fp32_grad, int fp32_numel,
                               const platform::float16 *fp16_grad,
                               int fp16_numel, float *nan_inf_flag,
                               gpuStream_t stream,
                               memory::Buffer *cub_tmp_buffer) {
  bool *fp32_has_nan_inf = nullptr;
  bool *fp16_has_nan_inf = nullptr;
  if (fp32_numel > 0) {
    fp32_has_nan_inf = reinterpret_cast<bool *>(nan_inf_flag + 1);
    hipcub::TransformInputIterator<bool, IsNanInfFunctor<float>, const float *>
    iter(fp32_grad, IsNanInfFunctor<float>());
    CubDeviceReduce(iter, fp32_has_nan_inf, fp32_numel, OrFunctor(), false,
                    stream, cub_tmp_buffer);
  }

  if (fp16_numel > 0) {
    fp16_has_nan_inf = reinterpret_cast<bool *>(nan_inf_flag + 1) + 1;
    hipcub::TransformInputIterator<bool, IsNanInfFunctor<platform::float16>,
                                const platform::float16 *>
        iter(fp16_grad, IsNanInfFunctor<platform::float16>());
    CubDeviceReduce(iter, fp16_has_nan_inf, fp16_numel, OrFunctor(), false,
                    stream, cub_tmp_buffer);
  }

  if (fp32_has_nan_inf && fp16_has_nan_inf) {
    SetNanInfValueCUDAKernelTwoFlag<<<1, 1, 0, stream>>>(
        fp32_has_nan_inf, fp16_has_nan_inf, nan_inf_flag);
  } else if (fp32_has_nan_inf) {
    SetNanInfValueCUDAKernelOneFlag<<<1, 1, 0, stream>>>(fp32_has_nan_inf,
                                                         nan_inf_flag);
  } else {
    SetNanInfValueCUDAKernelOneFlag<<<1, 1, 0, stream>>>(fp16_has_nan_inf,
                                                         nan_inf_flag);
  }
}

template <typename T>
static void FillZeroWithPtr(T *x, size_t n, gpuStream_t stream) {
  static_assert(!std::is_same<T, void>::value, "T cannot be void.");
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemsetAsync(x, 0, n * sizeof(T), stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemsetAsync(x, 0, n * sizeof(T), stream));
#endif
}

template <typename T>
class DistributedFusedLambOpKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    auto place = dev_ctx.GetPlace();

    // Step 1: Get fp16 param and grad tensors
    int64_t fp16_numel;
    auto *fp16_param = GetSameInOutTensorPtr<platform::float16, true>(
        ctx, place, "FP16FusedParam", "FP16FusedParamOut", &fp16_numel);
    bool has_fp16_param = (fp16_numel > 0);
    const platform::float16 *fp16_grad = nullptr;
    if (has_fp16_param) {
      fp16_grad = GetInputTensorPtr<platform::float16>(ctx, "FP16FusedGrad");
    } else {
      fp16_param = nullptr;
    }

    // Step 2: Get fp32 param and grad tensors
    int64_t fp32_numel = 0;
    auto *fp32_param = GetSameInOutTensorPtr<float, true>(
        ctx, place, "FP32FusedParam", "FP32FusedParamOut", &fp32_numel);
    PADDLE_ENFORCE_GE(fp32_numel, fp16_numel,
                      platform::errors::InvalidArgument(
                          "The element number in FP32FusedParam should be not "
                          "less than FP16FusedParam."));

    fp32_numel -= fp16_numel;  // the FP32FusedParam contains fp32 param and
                               // fp16 master weight
    bool has_fp32_param = (fp32_numel > 0);
    const float *fp32_grad = nullptr;
    if (has_fp32_param) {
      fp32_grad = GetInputTensorPtr<float>(ctx, "FP32FusedGrad");
    } else {
      PADDLE_ENFORCE_EQ(
          has_fp16_param, true,
          platform::errors::InvalidArgument(
              "Either FP32FusedGrad or FP16FusedGrad cannot be NULL."));
    }

    auto numel = fp32_numel + fp16_numel;
    VLOG(1) << "numel = " << numel << " , fp32_numel = " << fp32_numel
            << " , fp16_numel = " << fp16_numel;

    // The NVIDIA cub library does not support number > INT32_MAX
    PADDLE_ENFORCE_LE(numel, std::numeric_limits<int>::max(),
                      platform::errors::Unimplemented(
                          "Too many parameter number. Only <= %d is supported.",
                          std::numeric_limits<int>::max()));

    // Step 3: Get FusedIndices, ParamInfo
    const auto *indices = GetInputTensorPtr<int>(ctx, "FusedIndices");
    const auto *param_info_tensor = GetInputTensorPtr<int>(ctx, "ParamInfo");
    auto fp32_local_start_idx = param_info_tensor[0];
    auto fp32_local_param_num = param_info_tensor[1];
    auto fp32_global_param_num = param_info_tensor[2];
    auto fp16_local_start_idx = param_info_tensor[3];
    auto fp16_local_param_num = param_info_tensor[4];
    auto fp16_global_param_num = param_info_tensor[5];

    auto local_param_num = fp32_local_param_num + fp16_local_param_num;
    auto param_num = fp32_global_param_num + fp16_global_param_num;
    PADDLE_ENFORCE_LE(local_param_num, param_num,
                      platform::errors::InvalidArgument(
                          "The local parameter number should not exceed the "
                          "global parameter number."));
    VLOG(1) << "local_param_num = " << local_param_num
            << " , global_param_num = " << param_num
            << " , fp32_local_start_idx = " << fp32_local_start_idx
            << " , fp32_local_param_num = " << fp32_local_param_num
            << " , fp32_global_param_num = " << fp32_global_param_num
            << " , fp16_local_start_idx = " << fp16_local_start_idx
            << " , fp16_local_param_num = " << fp16_local_param_num
            << " , fp16_global_param_num = " << fp16_global_param_num;

    // Step 4: Get LearningRate, Moment1, Moment2, Beta1Pow, Beta2Pow,
    // WeightDecay, GlobalScale, FoundInf
    const auto *global_scale = GetInputTensorPtr<float>(ctx, "GlobalScale");
    const auto *lr = GetInputTensorPtr<float>(ctx, "LearningRate");
    int64_t partial_numel = 0;
    auto *moment1 = GetSameInOutTensorPtr<float>(ctx, place, "Moment1",
                                                 "Moment1Out", &partial_numel);

    PADDLE_ENFORCE_EQ(numel % partial_numel, 0,
                      platform::errors::InvalidArgument(
                          "The total parameter number %d should be divided "
                          "exactly by the element number %d of Moment1.",
                          numel, partial_numel));

    int64_t num_devices = numel / partial_numel;
    VLOG(1) << "num_devices = " << num_devices
            << " , partial_numel = " << partial_numel;

    PADDLE_ENFORCE_EQ(fp32_numel % num_devices, 0,
                      platform::errors::InvalidArgument(
                          "The fp32 parameter number %d should be divided "
                          "exactly by the device number %d.",
                          fp32_numel, num_devices));
    PADDLE_ENFORCE_EQ(fp16_numel % num_devices, 0,
                      platform::errors::InvalidArgument(
                          "The fp16 parameter number %d should be divided "
                          "exactly by the device number %d.",
                          fp16_numel, num_devices));

    auto *moment2 =
        GetSameInOutTensorPtr<float>(ctx, place, "Moment2", "Moment2Out");
    auto *beta1pow =
        GetSameInOutTensorPtr<float>(ctx, place, "Beta1Pow", "Beta1PowOut");
    auto *beta2pow =
        GetSameInOutTensorPtr<float>(ctx, place, "Beta2Pow", "Beta2PowOut");
    const float *weight_decay = GetInputTensorPtr<float>(ctx, "WeightDecay");

    auto *found_inf_t = ctx.Output<framework::Tensor>("FoundInf");
    found_inf_t->Resize({1});
    auto *found_inf = found_inf_t->mutable_data<bool>(place);

    // Step 5: Get attributes beta1, beta2, epsilon, max_grad_norm, ring_id,
    // use_master_param_norm, is_grad_scaled_by_nranks
    auto beta1 = ctx.Attr<float>("beta1");
    auto beta2 = ctx.Attr<float>("beta2");
    auto epsilon = ctx.Attr<float>("epsilon");
    auto max_global_grad_norm = ctx.Attr<float>("max_global_grad_norm");
    auto clip_after_allreduce = ctx.Attr<bool>("clip_after_allreduce");
    auto ring_id = ctx.Attr<int>("ring_id");
    auto use_master_param_norm = ctx.Attr<bool>("use_master_param_norm");
    auto is_grad_scaled_by_nranks = ctx.Attr<bool>("is_grad_scaled_by_nranks");
    VLOG(10) << "max_global_grad_norm = " << max_global_grad_norm
             << " , clip_after_allreduce = " << clip_after_allreduce
             << " , use_master_param_norm = " << use_master_param_norm
             << " , is_grad_scaled_by_nranks = " << is_grad_scaled_by_nranks;

    // Step 6: allreduce + global norm gradient clip
    int rank = 0;
    ncclComm_t comm = nullptr;
    if (num_devices > 1) {
      auto *nccl_comm_handle =
          platform::NCCLCommContext::Instance().Get(ring_id, place);
      comm = nccl_comm_handle->comm();
      rank = nccl_comm_handle->rank();
    }

    memory::Buffer grad_norm_square_buffer(place);
    auto *fp32_square_grad_norm = grad_norm_square_buffer.Alloc<float>(2);
    memory::Buffer cub_tmp_buffer(place);

    memory::Buffer sum_grad_buffer(place);
    float *fp32_sum_grad;
    platform::float16 *fp16_sum_grad;
    auto fp32_numel_each_device = fp32_numel / num_devices;
    auto fp16_numel_each_device = fp16_numel / num_devices;
    if (num_devices > 1) {
      auto ptr = sum_grad_buffer.Alloc<uint8_t>(
          fp32_numel_each_device * sizeof(float) +
          fp16_numel_each_device * sizeof(platform::float16));
      fp32_sum_grad = has_fp32_param ? reinterpret_cast<float *>(ptr) : nullptr;
      fp16_sum_grad = has_fp16_param
                          ? reinterpret_cast<platform::float16 *>(
                                ptr + fp32_numel_each_device * sizeof(float))
                          : nullptr;
    } else {
      // NOTE: The const_cast here is not important. The fp32_sum_grad and
      // fp16_sum_grad would not be changed when num_devices == 1
      // But if I do not perform const_cast here, there would be more
      // if-else codes (num_devices > 1) when I write the following code.
      // So I prefer to use const_cast to unify the following code to reduce
      // the if-else codes.
      fp32_sum_grad = const_cast<float *>(fp32_grad);
      fp16_sum_grad = const_cast<platform::float16 *>(fp16_grad);
    }

    float rescale_grad = 1.0f;
    if (!is_grad_scaled_by_nranks) {
      rescale_grad /= num_devices;
    }

    if (max_global_grad_norm > 0) {
      if (clip_after_allreduce) {
        // (1) ReduceScater first
        NCCLReduceScatterWithScale(fp32_grad, fp32_sum_grad,
                                   fp32_numel_each_device, num_devices, comm,
                                   stream, dev_ctx);
        NCCLReduceScatterWithScale(fp16_grad, fp16_sum_grad,
                                   fp16_numel_each_device, num_devices, comm,
                                   stream, dev_ctx);
        // (2) Calculate the global grad norm
        GetSquareGradNorm(fp32_sum_grad, fp32_numel_each_device, fp16_sum_grad,
                          fp16_numel_each_device, fp32_square_grad_norm, stream,
                          &cub_tmp_buffer);
        VLOG(1) << "Grad square norm before all reduce: "
                << FlattenToString(fp32_square_grad_norm, 1, place);
        if (num_devices > 1) {
          PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
              fp32_square_grad_norm, fp32_square_grad_norm, 1, ncclFloat32,
              ncclSum, comm, stream));
        }
        VLOG(1) << "Grad square norm after all reduce: "
                << FlattenToString(fp32_square_grad_norm, 1, place);
      } else {
        // (1) Calculate the local grad norm
        GetSquareGradNorm(fp32_grad, fp32_numel, fp16_grad, fp16_numel,
                          fp32_square_grad_norm, stream, &cub_tmp_buffer);
        VLOG(1) << "Grad square norm before all reduce: "
                << FlattenToString(fp32_square_grad_norm, 1, place);
        // (2) Calculate the gradient clip scale
        float *fp32_scale = nullptr;
        platform::float16 *fp16_scale = nullptr;
        if (has_fp32_param && has_fp16_param) {
          auto *ptr = cub_tmp_buffer.Alloc<uint8_t>(sizeof(float) +
                                                    sizeof(platform::float16));
          fp32_scale = reinterpret_cast<float *>(ptr);
          fp16_scale =
              reinterpret_cast<platform::float16 *>(ptr + sizeof(float));
        } else if (has_fp32_param) {
          fp32_scale = cub_tmp_buffer.Alloc<float>(1);
        } else {
          fp16_scale = cub_tmp_buffer.Alloc<platform::float16>(1);
        }

        float clip_scale = 1.0f;
        if (is_grad_scaled_by_nranks) {
          clip_scale *= num_devices;
        }
        CalcGradNormClipBeforeAllReduceScale<
            float, platform::float16><<<1, 1, 0, stream>>>(
            global_scale, max_global_grad_norm, fp32_square_grad_norm,
            fp32_scale, fp16_scale, clip_scale);
        VLOG(1) << "Grad scale: " << FlattenToString(fp32_scale, 1, place);
        if (num_devices > 1) {
          PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
              fp32_square_grad_norm, fp32_square_grad_norm, 1, ncclFloat32,
              ncclSum, comm, stream));
        }
        // (3) Do ReduceScatter with scale
        NCCLReduceScatterWithScale(fp32_grad, fp32_sum_grad,
                                   fp32_numel_each_device, num_devices, comm,
                                   stream, dev_ctx, fp32_scale);
        NCCLReduceScatterWithScale(fp16_grad, fp16_sum_grad,
                                   fp16_numel_each_device, num_devices, comm,
                                   stream, dev_ctx, fp16_scale);
        // (4) mark max_global_grad_norm as 0, meaning that clip has been
        // already performed
        max_global_grad_norm = 0;
      }
    } else {
      NCCLReduceScatterWithScale(fp32_grad, fp32_sum_grad,
                                 fp32_numel_each_device, num_devices, comm,
                                 stream, dev_ctx);
      NCCLReduceScatterWithScale(fp16_grad, fp16_sum_grad,
                                 fp16_numel_each_device, num_devices, comm,
                                 stream, dev_ctx);
      CheckHasNanInfGrad(fp32_sum_grad, fp32_numel_each_device, fp16_sum_grad,
                         fp16_numel_each_device, fp32_square_grad_norm, stream,
                         &cub_tmp_buffer);
      if (num_devices > 1) {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
            fp32_square_grad_norm, fp32_square_grad_norm, 1, ncclFloat32,
            ncclSum, comm, stream));
      }
      max_global_grad_norm = 0;
    }
    VLOG(10) << "ReduceScatter done";

    // Step 7: update the moment1, moment2. Calcuate the trust_ratio_div
    memory::Buffer trust_ratio_div_buffer(place);
    auto *trust_ratio_div = trust_ratio_div_buffer.Alloc<float>(partial_numel);
    auto fp32_offset = rank * fp32_numel_each_device;
    auto fp16_offset = rank * fp16_numel_each_device;
    if (has_fp32_param) {
      auto config =
          platform::GetGpuLaunchConfig1D(dev_ctx, fp32_numel_each_device);
      VLOG(10) << "Update FP32 Moment and TrustRatioDiv starts";
      UpdateLambMoment<<<config.block_per_grid, config.thread_per_block, 0,
                         stream>>>(
          fp32_param + fp32_offset, fp32_sum_grad, fp32_square_grad_norm,
          global_scale, indices + fp32_offset, weight_decay, beta1pow, beta2pow,
          moment1, moment2, trust_ratio_div, beta1, beta2, epsilon,
          max_global_grad_norm, fp32_numel_each_device, rescale_grad);
      VLOG(10) << "Update FP32 Moment and TrustRatioDiv done";
    }
    float *master_param = nullptr;
    if (has_fp16_param) {
      master_param = fp32_param + fp32_numel;
      auto config =
          platform::GetGpuLaunchConfig1D(dev_ctx, fp16_numel_each_device);
      VLOG(10) << "Update FP16 Moment and TrustRatioDiv starts";
      UpdateLambMoment<<<config.block_per_grid, config.thread_per_block, 0,
                         stream>>>(
          master_param + fp16_offset, fp16_sum_grad, fp32_square_grad_norm,
          global_scale, indices + fp32_numel + fp16_offset, weight_decay,
          beta1pow, beta2pow, moment1 + fp32_numel_each_device,
          moment2 + fp32_numel_each_device,
          trust_ratio_div + fp32_numel_each_device, beta1, beta2, epsilon,
          max_global_grad_norm, fp16_numel_each_device, rescale_grad);
      VLOG(10) << "Update FP16 Moment and TrustRatioDiv done";
    }

    VLOG(10) << "Update Moment and TrustRatioDiv done hehahaha";

    // Step 8: calculate L2-Norm square of parameter and trust_ratio_div
    memory::Buffer square_norm_buffer(place);
    auto *param_square_norm = square_norm_buffer.Alloc<float>(2 * param_num);
    auto *trust_ratio_div_square_norm = param_square_norm + param_num;

    auto *fused_offsets_t = ctx.Input<framework::Tensor>("FusedParamOffsets");
    auto *fused_offsets = fused_offsets_t->data<int>();
    auto *fp32_partial_fused_offsets_t =
        ctx.Input<framework::Tensor>("FP32PartialFusedParamOffsets");
    const auto *fp32_partial_fused_offsets =
        fp32_partial_fused_offsets_t->data<int>();
    auto *fp16_partial_fused_offsets_t =
        ctx.Input<framework::Tensor>("FP16PartialFusedParamOffsets");
    const auto *fp16_partial_fused_offsets =
        fp16_partial_fused_offsets_t->data<int>();

    VLOG(1) << "FusedParamOffsets: "
            << FlattenToString(fused_offsets, fused_offsets_t->numel(), place);
    VLOG(1) << "FP32PartialFusedParamOffsets: "
            << FlattenToString(fp32_partial_fused_offsets,
                               fp32_partial_fused_offsets_t->numel(), place);
    VLOG(1) << "FP16PartialFusedParamOffsets: "
            << FlattenToString(fp16_partial_fused_offsets,
                               fp16_partial_fused_offsets_t->numel(), place);

    if (num_devices > 1) {
      if (use_master_param_norm) {
        FillZeroWithPtr(param_square_norm + fp32_global_param_num,
                        2 * param_num - fp32_global_param_num, stream);
      } else {
        FillZeroWithPtr(trust_ratio_div_square_norm, param_num, stream);
      }
    }
    CubDeviceSegmentedSquareNorm(fp32_param, param_square_norm,
                                 fp32_global_param_num, fused_offsets, 0,
                                 stream, &cub_tmp_buffer);
    if (use_master_param_norm) {
      CubDeviceSegmentedSquareNorm(
          master_param + fp16_offset, param_square_norm + fp16_local_start_idx,
          fp16_local_param_num, fp16_partial_fused_offsets, 0, stream,
          &cub_tmp_buffer);
    } else {
      // NOTE: extra computation is performed. We can improve this performance
      // if needed in the future.
      CubDeviceSegmentedSquareNorm(
          fp16_param, param_square_norm + fp32_global_param_num,
          fp16_global_param_num, fused_offsets + fp32_global_param_num,
          static_cast<int>(fp32_numel), stream, &cub_tmp_buffer);
    }

    CubDeviceSegmentedSquareNorm(
        trust_ratio_div, trust_ratio_div_square_norm + fp32_local_start_idx,
        fp32_local_param_num, fp32_partial_fused_offsets, 0, stream,
        &cub_tmp_buffer);
    CubDeviceSegmentedSquareNorm(
        trust_ratio_div + fp32_numel_each_device,
        trust_ratio_div_square_norm + fp16_local_start_idx,
        fp16_local_param_num, fp16_partial_fused_offsets, 0, stream,
        &cub_tmp_buffer);

    VLOG(1) << "TrustRatioDiv L2-Norm before allreduce: "
            << FlattenToString(trust_ratio_div_square_norm, param_num, place);
    if (num_devices > 1) {
      if (use_master_param_norm) {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
            param_square_norm + fp32_global_param_num,
            param_square_norm + fp32_global_param_num,
            2 * param_num - fp32_global_param_num, ncclFloat32, ncclSum, comm,
            stream));
      } else {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
            trust_ratio_div_square_norm, trust_ratio_div_square_norm, param_num,
            ncclFloat32, ncclSum, comm, stream));
      }
      VLOG(10) << "ncclAllReduce done";
    }

    LogParamAndTrustRatioDivSquareNorm<1>(ctx, param_square_norm,
                                          trust_ratio_div_square_norm);
    VLOG(10) << "Calculate L2-Norm of Param and TrustRatioDiv done";

    // Step 9: update parameter, beta1pow, beta2pow. All gather parameters.
    if (has_fp32_param) {
      LambUpdateParamAndBetaPows<float>(
          dev_ctx, trust_ratio_div, lr, indices + fp32_offset,
          param_square_norm, trust_ratio_div_square_norm, fp32_square_grad_norm,
          &beta1pow, &beta2pow, &found_inf, beta1, beta2,
          fp32_numel_each_device, fp32_param + fp32_offset, nullptr, stream);
      if (num_devices > 1) {
        // ncclAllGather
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllGather(
            fp32_param + fp32_offset, fp32_param, fp32_numel_each_device,
            ncclFloat32, comm, stream));
      }
    }
    if (has_fp16_param) {
      LambUpdateParamAndBetaPows<platform::float16>(
          dev_ctx, trust_ratio_div + fp32_numel_each_device, lr,
          indices + fp32_numel + fp16_offset, param_square_norm,
          trust_ratio_div_square_norm, fp32_square_grad_norm, &beta1pow,
          &beta2pow, &found_inf, beta1, beta2, fp16_numel_each_device,
          fp16_param + fp16_offset, master_param + fp16_offset, stream);

      if (num_devices > 1) {
        // ncclAllGather
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllGather(
            fp16_param + fp16_offset, fp16_param, fp16_numel_each_device,
            ncclFloat16, comm, stream));
      }
    }
    VLOG(10) << "Update Param done";

    VLOG(1) << "IsFinite: " << IsFinite(dev_ctx, fp32_square_grad_norm);
#else
    PADDLE_THROW(platform::errors::Unimplemented(
        "distributed_fused_lamb op should be used with NCCL/RCCL."));
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace plat = paddle::platform;
namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    distributed_fused_lamb,
    ops::DistributedFusedLambOpKernel<plat::CUDADeviceContext, float>);
