#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cmath>
#include "paddle/fluid/memory/buffer.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/optimizers/cast_with_ptr.h"
#include "paddle/fluid/operators/optimizers/distributed_fused_lamb_op.h"
#include "paddle/fluid/operators/optimizers/multi_tensor_apply.h"
#include "paddle/fluid/operators/tensor_to_string.h"
#include "paddle/fluid/platform/aligned_vector.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/for_range.h"
#include "paddle/fluid/string/string_helper.h"
#include "paddle/phi/core/utils/data_type.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#include "math.h"  // NOLINT
#endif

#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
#include "math.h"  // NOLINT
namespace cub = hipcub;
#endif

namespace paddle {
namespace operators {

template <typename T>
using MasterT = typename details::MPTypeTrait<T>::Type;

template <typename T>
static void FillZeroWithPtr(T *x, size_t n, gpuStream_t stream) {
  static_assert(!std::is_same<T, void>::value, "T cannot be void.");
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemsetAsync(x, 0, n * sizeof(T), stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemsetAsync(x, 0, n * sizeof(T), stream));
#endif
}

template <typename T, int BlockDim, int VecSize>
struct L2NormFunctor {
  DEVICE void operator()(int tensor_id, int chunk_id, int offset, int size,
                         const T *x, MasterT<T> *y, int max_chunk_num) const {
    using MT = MasterT<T>;
    const T *ptr = x + offset;

    using BlockReduce = hipcub::BlockReduce<MT, BlockDim>;
    __shared__ typename BlockReduce::TempStorage storage;

    MT square_sum = static_cast<MT>(0);
    int i;
    for (i = threadIdx.x * VecSize; i + VecSize <= size;
         i += (BlockDim * VecSize)) {
      platform::AlignedVector<T, VecSize> tmp_vec;
      platform::Load(ptr + i, &tmp_vec);
#pragma unroll
      for (int j = 0; j < VecSize; ++j) {
        auto tmp = static_cast<MT>(tmp_vec[j]);
        square_sum += (tmp * tmp);
      }
    }

    for (; i < size; ++i) {
      auto tmp = static_cast<MT>(ptr[i]);
      square_sum += (tmp * tmp);
    }

    square_sum = BlockReduce(storage).Reduce(square_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      y[tensor_id * max_chunk_num + chunk_id] = square_sum;
    }
  }
};

template <typename InT, typename OutT, int BlockDim>
static __global__ void MultiTensorL2NormReduceAgainCUDAKernel(
    const InT *x, OutT *y, int max_chunk_num) {
  int tensor_id = blockIdx.x;
  x += (tensor_id * max_chunk_num);
  using BlockReduce = hipcub::BlockReduce<InT, BlockDim>;
  __shared__ typename BlockReduce::TempStorage storage;
  InT sum = static_cast<InT>(0);
  for (int i = threadIdx.x; i < max_chunk_num; i += BlockDim) {
    sum += x[i];
  }
  sum = BlockReduce(storage).Reduce(sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    y[blockIdx.x] = static_cast<OutT>(sum);
  }
}

template <typename T>
static int GetChunkedVecSize(const T *ptr, int chunk_size) {
  static_assert(!std::is_same<T, void>::value, "T cannot be void.");

  constexpr int max_load_bits = 128;
  int valid_vec_size = max_load_bits / CHAR_BIT / sizeof(T);
  auto address = reinterpret_cast<uintptr_t>(ptr);
  constexpr int vec8 = alignof(platform::AlignedVector<T, 8>);
  constexpr int vec4 = alignof(platform::AlignedVector<T, 4>);
  constexpr int vec2 = alignof(platform::AlignedVector<T, 2>);
  chunk_size *= sizeof(T);
  if (address % vec8 == 0 && chunk_size % vec8 == 0) {
    return std::min(8, valid_vec_size);
  } else if (address % vec4 == 0 && chunk_size % vec4 == 0) {
    return std::min(4, valid_vec_size);
  } else if (address % vec2 == 0 && chunk_size % vec2 == 0) {
    return std::min(2, valid_vec_size);
  } else {
    return 1;
  }
}

#define PD_VEC_LAUNCH_KERNEL_CASE(__vec_size, ...) \
  case __vec_size: {                               \
    constexpr int kVecSize = __vec_size;           \
    __VA_ARGS__;                                   \
    break;                                         \
  }

#define PD_VEC_LAUNCH_KERNEL(__vec_size, ...)    \
  do {                                           \
    switch (__vec_size) {                        \
      PD_VEC_LAUNCH_KERNEL_CASE(8, __VA_ARGS__); \
      PD_VEC_LAUNCH_KERNEL_CASE(4, __VA_ARGS__); \
      PD_VEC_LAUNCH_KERNEL_CASE(2, __VA_ARGS__); \
      PD_VEC_LAUNCH_KERNEL_CASE(1, __VA_ARGS__); \
    }                                            \
  } while (0)

// TODO(zengjinle): which chunk_size is better?
template <typename InT, typename OutT, int MaxTensorNumPerLaunch = 160,
          int MaxChunkNumPerLaunch = 780>
static void MultiTensorL2Norm(const platform::CUDAPlace &place,
                              gpuStream_t stream, const InT *x,
                              const int *offsets, int n, OutT *y,
                              int chunk_size = 65536) {
  if (n <= 0) return;

  constexpr int kNumTensor = MaxTensorNumPerLaunch;
  constexpr int kNumChunk = MaxChunkNumPerLaunch;
  constexpr int kBlockDim = 512;

  int max_chunk_num = -1;
  int vec_size = 8;
  int total_chunk_num = 0;
  for (int i = 0; i < n; ++i) {
    vec_size = std::min(
        vec_size, GetChunkedVecSize(x + offsets[i] - offsets[0], chunk_size));
    int length = offsets[i + 1] - offsets[i];
    auto tmp_chunk_num = (length + chunk_size - 1) / chunk_size;
    max_chunk_num = std::max(max_chunk_num, tmp_chunk_num);
    total_chunk_num += tmp_chunk_num;
  }

  VLOG(1) << "MultiTensorL2Norm max_chunk_num = " << max_chunk_num
          << " , total_chunk_num = " << total_chunk_num
          << " , tensor_num = " << n;

  using MT = MasterT<InT>;
  memory::Buffer tmp_out(place);
  auto *tmp_out_ptr = tmp_out.Alloc<MT>(n * max_chunk_num);
  FillZeroWithPtr(tmp_out_ptr, n * max_chunk_num, stream);

#define PD_LAUNCH_MULTI_TENSOR_APPLY_L2_NORM_KERNEL                            \
  do {                                                                         \
    using FunctorT = L2NormFunctor<InT, kBlockDim, kVecSize>;                  \
    VLOG(10) << __func__ << " " << typeid(InT).name()                          \
             << " VecSize = " << kVecSize;                                     \
    MultiTensorApply<FunctorT, kNumTensor, kNumChunk>(                         \
        FunctorT(), stream, offsets, n, chunk_size, kBlockDim, x, tmp_out_ptr, \
        max_chunk_num);                                                        \
  } while (0)

  PD_VEC_LAUNCH_KERNEL(vec_size, PD_LAUNCH_MULTI_TENSOR_APPLY_L2_NORM_KERNEL);
#undef PD_LAUNCH_MULTI_TENSOR_APPLY_L2_NORM_KERNEL

  MultiTensorL2NormReduceAgainCUDAKernel<
      MT, OutT, kBlockDim><<<n, kBlockDim, 0, stream>>>(tmp_out_ptr, y,
                                                        max_chunk_num);
}

template <int LogLevel>
static void LogParamAndTrustRatioDivSquareNorm(
    const framework::ExecutionContext &ctx, const float *param_square_norm,
    const float *trust_ratio_div_square_norm) {
  if (!VLOG_IS_ON(LogLevel)) return;

  auto tensors = ctx.MultiInput<framework::Tensor>("Param");
  if (tensors.empty()) return;

  const auto *order = ctx.Input<framework::Tensor>("ParamOrder")->data<int>();

  size_t n = tensors.size();
  auto place = tensors[0]->place();

  auto pn_vec = ToVector(param_square_norm, n, place);
  auto tn_vec = ToVector(trust_ratio_div_square_norm, n, place);

  const auto &names = ctx.GetOp().Inputs("Param");
  for (size_t i = 0; i < n; ++i) {
    auto idx = order[i];
    VLOG(LogLevel) << "Param " << tensors[idx]->dtype() << " " << names[idx]
                   << " pn = " << pn_vec[i] << " , tn = " << tn_vec[i];
  }
}

static bool IsFinite(const platform::CUDADeviceContext &dev_ctx,
                     const float *ptr) {
  auto stream = dev_ctx.stream();
  float cpu_value;
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&cpu_value, ptr, sizeof(float),
                                            hipMemcpyDeviceToHost, stream));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&cpu_value, ptr, sizeof(float),
                                             hipMemcpyDeviceToHost, stream));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#endif
  LOG(INFO) << "NAN_INF indicator value: " << cpu_value;
  return isfinite(cpu_value);
}

template <typename T>
static const T *GetInputTensorPtr(const framework::ExecutionContext &ctx,
                                  const char *in_name,
                                  int64_t *numel = nullptr) {
  const auto *in_tensor = ctx.Input<framework::Tensor>(in_name);
  PADDLE_ENFORCE_NOT_NULL(in_tensor, platform::errors::InvalidArgument(
                                         "Input(%s) cannot be NULL.", in_name));
  if (in_tensor->IsInitialized()) {
    if (numel) *numel = in_tensor->numel();
    return in_tensor->data<T>();
  } else {
    if (numel) *numel = 0;
    return nullptr;
  }
}

template <typename T, bool AllowNotExist = false>
static T *GetSameInOutTensorPtr(const framework::ExecutionContext &ctx,
                                const platform::Place &place,
                                const char *in_name, const char *out_name,
                                int64_t *numel = nullptr) {
  const auto *in_tensor = ctx.Input<framework::Tensor>(in_name);
  if (in_tensor == nullptr || !in_tensor->IsInitialized()) {
    PADDLE_ENFORCE_EQ(AllowNotExist, true,
                      platform::errors::InvalidArgument(
                          "Input(%s) cannot be NULL.", in_name));
    if (numel) *numel = 0;
    return nullptr;
  }

  auto *out_tensor = ctx.Output<framework::Tensor>(out_name);
  PADDLE_ENFORCE_NOT_NULL(in_tensor, platform::errors::InvalidArgument(
                                         "Input(%s) cannot be NULL.", in_name));
  PADDLE_ENFORCE_NOT_NULL(out_tensor,
                          platform::errors::InvalidArgument(
                              "Output(%s) cannot be NULL.", out_name));
  const T *in_data = in_tensor->data<T>();
  T *out_data = out_tensor->mutable_data<T>(place);
  PADDLE_ENFORCE_EQ(in_data, out_data,
                    platform::errors::InvalidArgument(
                        "Input(%s) and Output(%s) must be the same Tensor.",
                        in_name, out_name));
  if (numel) *numel = out_tensor->numel();
  return out_data;
}

template <typename T>
struct SquareFunctor {
  HOSTDEVICE MasterT<T> operator()(T x) const {
    auto y = static_cast<MasterT<T>>(x);
    return y * y;
  }
};

template <typename T>
struct IsNanInfFunctor {
  HOSTDEVICE bool operator()(T x) const { return !isfinite(x); }
};

struct OrFunctor {
  HOSTDEVICE bool operator()(bool x, bool y) const { return x || y; }
};

struct AndFunctor {
  HOSTDEVICE bool operator()(bool x, bool y) const { return x && y; }
};

template <typename T1, typename T2, int VecSize>
static __global__ void ScaleCUDAKernel(const T1 *__restrict__ x,
                                       const T2 *__restrict__ scale,
                                       T1 *__restrict__ y, int num) {
  static_assert(sizeof(T1) <= sizeof(T2),
                "sizeof(T1) must be not greater than sizeof(T2).");
  T2 s = scale[0];

  int i = (threadIdx.x + blockIdx.x * blockDim.x) * VecSize;
  int stride = blockDim.x * gridDim.x * VecSize;

  for (; i + VecSize <= num; i += stride) {
    platform::AlignedVector<T1, VecSize> x_vec;
    platform::AlignedVector<T1, VecSize> y_vec;

    platform::Load(x + i, &x_vec);
#pragma unroll
    for (int j = 0; j < VecSize; ++j) {
      y_vec[j] = static_cast<T1>(static_cast<T2>(x_vec[j]) * s);
    }
    platform::Store(y_vec, y + i);
  }

  for (; i < num; ++i) {
    y[i] = static_cast<T1>(static_cast<T2>(x[i]) * s);
  }
}

template <typename T>
static __global__ void AddToCUDAKernel(const T *__restrict__ x,
                                       T *__restrict__ y) {
  y[0] += x[0];
}

// If clip before allreduce,
// coeff = global_scale * max_global_grad_norm / (1e-6 + sqrt(square_grad_norm)
// * rescale_grad)
// if coeff >= 1 or coeff is Nan/Inf, scale = 1.0
// else scale = coeff
template <typename T1, typename T2>
static __global__ void CalcGradNormClipBeforeAllReduceScale(
    const T1 *__restrict__ global_scale, T1 max_global_grad_norm,
    const T1 *__restrict__ square_grad_norm, T1 *__restrict__ out1,
    T2 *__restrict__ out2, T1 clip_rescale_grad) {
  T1 grad_norm = static_cast<T1>(sqrtf(*square_grad_norm)) * clip_rescale_grad;
  T1 scale = global_scale[0] * max_global_grad_norm / (1e-6 + grad_norm);
  bool found_nan_inf = !isfinite(scale);
  if (scale >= 1 || found_nan_inf) {
    scale = static_cast<T1>(1.0);
  }

  if (out1) {
    *out1 = scale;
  }
  if (out2) {
    *out2 = static_cast<T2>(scale);
  }
}

static __global__ void SetNanInfValueCUDAKernelOneFlag(const bool *in_flag_p,
                                                       float *out_p) {
  *out_p = (*in_flag_p) ? __int_as_float(0x7fffffffU) : 0.0f;
}

static __global__ void SetNanInfValueCUDAKernelTwoFlag(const bool *in_flag_p_1,
                                                       const bool *in_flag_p_2,
                                                       float *out_p) {
  *out_p =
      ((*in_flag_p_1) || (*in_flag_p_2)) ? __int_as_float(0x7fffffffU) : 0.0f;
}

template <typename T, typename GradT, int VecSize>
static __global__ void UpdateLambMomentAndTrustRatioDivCUDAKernel(
    const T *__restrict__ param_p, const GradT *__restrict__ grad_p,
    const T *__restrict__ square_grad_norm_p,
    const T *__restrict__ global_scale, const T *__restrict__ beta1pow_p,
    const T *__restrict__ beta2pow_p, T *__restrict__ mom1_p,
    T *__restrict__ mom2_p, T *__restrict__ trust_ratio_div_p, bool *found_inf,
    T weight_decay, int weight_decay_end_numel, T beta1, T beta2, T epsilon,
    T max_global_grad_norm, int num, T rescale_grad) {
  T square_grad_norm = *square_grad_norm_p;
  bool need_update_found_inf =
      (found_inf && threadIdx.x == 0 && blockIdx.x == 0);
  if (!isfinite(square_grad_norm)) {
    if (need_update_found_inf) *found_inf = true;
    return;
  } else if (need_update_found_inf) {
    *found_inf = false;
  }

  T scale = rescale_grad / global_scale[0];
  if (max_global_grad_norm > 0) {
    T clip_scale =
        max_global_grad_norm / (sqrtf(square_grad_norm) * scale + 1e-6);
    if (clip_scale < static_cast<T>(1)) {
      scale *= clip_scale;
    }
  }

  T one_minus_beta1pow = 1 - beta1pow_p[0];
  T one_minus_beta2pow = 1 - beta2pow_p[0];

  int i = (threadIdx.x + blockIdx.x * blockDim.x) * VecSize;
  int stride = blockDim.x * gridDim.x * VecSize;

  for (; i + VecSize <= num; i += stride) {
    platform::AlignedVector<T, VecSize> param_vec;
    platform::AlignedVector<GradT, VecSize> grad_vec;
    platform::AlignedVector<T, VecSize> mom1_vec;
    platform::AlignedVector<T, VecSize> mom2_vec;
    platform::AlignedVector<T, VecSize> trust_ratio_div_vec;

    T cur_weight_decay = (i < weight_decay_end_numel) * weight_decay;
    if (cur_weight_decay != static_cast<T>(0.0)) {
      platform::Load(param_p + i, &param_vec);
    } else {
#pragma unroll
      for (int j = 0; j < VecSize; ++j) {
        param_vec[j] = static_cast<T>(0);
      }
    }
    platform::Load(grad_p + i, &grad_vec);
    platform::Load(mom1_p + i, &mom1_vec);
    platform::Load(mom2_p + i, &mom2_vec);

#define PD_LAMB_MOM_TRUST_RATIO_DIV_UPDATE(__param, __grad, __mom1, __mom2,    \
                                           __trust_ratio_div, __idx)           \
  T p = __param[__idx];                                                        \
  T g = static_cast<T>(__grad[__idx]) * scale;                                 \
  T mom1 = __mom1[__idx];                                                      \
  T mom2 = __mom2[__idx];                                                      \
  mom1 = beta1 * mom1 + (1 - beta1) * g;                                       \
  mom2 = beta2 * mom2 + (1 - beta2) * g * g;                                   \
  T mom1_unbiased = mom1 / one_minus_beta1pow;                                 \
  T mom2_unbiased = mom2 / one_minus_beta2pow;                                 \
  __trust_ratio_div[__idx] =                                                   \
      mom1_unbiased / (sqrtf(mom2_unbiased) + epsilon) + cur_weight_decay * p; \
  __mom1[__idx] = mom1;                                                        \
  __mom2[__idx] = mom2;

#pragma unroll
    for (int j = 0; j < VecSize; ++j) {
      PD_LAMB_MOM_TRUST_RATIO_DIV_UPDATE(param_vec, grad_vec, mom1_vec,
                                         mom2_vec, trust_ratio_div_vec, j);
    }

    platform::Store(mom1_vec, mom1_p + i);
    platform::Store(mom2_vec, mom2_p + i);
    platform::Store(trust_ratio_div_vec, trust_ratio_div_p + i);
  }

  for (; i < num; ++i) {
    T cur_weight_decay = (i < weight_decay_end_numel) * weight_decay;
    PD_LAMB_MOM_TRUST_RATIO_DIV_UPDATE(param_p, grad_p, mom1_p, mom2_p,
                                       trust_ratio_div_p, i);
  }
}

template <typename T, typename GradT>
static void MultiTensorUpdateLambMomentAndTrustRatioDiv(
    const platform::CUDADeviceContext &dev_ctx, const int *offsets, int n,
    const T *param_p, const GradT *grad_p, const T *square_grad_norm_p,
    const T *global_scale, const T *beta1pow_p, const T *beta2pow_p, T *mom1_p,
    T *mom2_p, T *trust_ratio_div_p, bool *found_inf_p, T weight_decay,
    int weight_decay_end_idx, T beta1, T beta2, T epsilon,
    T max_global_grad_norm, T rescale_grad) {
  if (n <= 0) return;
  int numel = offsets[n] - offsets[0];
  PADDLE_ENFORCE_GE(weight_decay_end_idx, 0,
                    platform::errors::InvalidArgument(
                        "The weight decay end index should be >= 0."));
  PADDLE_ENFORCE_LE(weight_decay_end_idx, n,
                    platform::errors::InvalidArgument(
                        "The weight decay end index should be < %d.", n));
  auto weight_decay_end_numel = offsets[weight_decay_end_idx] - offsets[0];

  int vec_size = GetChunkedVecSize(param_p, 0);
  vec_size = std::min(vec_size, GetChunkedVecSize(grad_p, 0));
  vec_size = std::min(vec_size, GetChunkedVecSize(mom1_p, 0));
  vec_size = std::min(vec_size, GetChunkedVecSize(mom2_p, 0));
  vec_size = std::min(vec_size, GetChunkedVecSize(trust_ratio_div_p, 0));
  for (int i = 0; i < n; ++i) {
    auto length = offsets[i + 1] - offsets[i];
    while (length % vec_size != 0) {
      vec_size /= 2;
    }
  }

  VLOG(1) << __func__ << " VecSize = " << vec_size;

  auto stream = dev_ctx.stream();
  auto config = platform::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

#define PD_LAUNCH_LAMB_MOM_TRUST_RATIO_DIV_KERNEL                      \
  do {                                                                 \
    UpdateLambMomentAndTrustRatioDivCUDAKernel<T, GradT, kVecSize><<<  \
        config.block_per_grid, config.thread_per_block, 0, stream>>>(  \
        param_p, grad_p, square_grad_norm_p, global_scale, beta1pow_p, \
        beta2pow_p, mom1_p, mom2_p, trust_ratio_div_p, found_inf_p,    \
        weight_decay, weight_decay_end_numel, beta1, beta2, epsilon,   \
        max_global_grad_norm, numel, rescale_grad);                    \
  } while (0)

  PD_VEC_LAUNCH_KERNEL(vec_size, PD_LAUNCH_LAMB_MOM_TRUST_RATIO_DIV_KERNEL);
#undef PD_LAUNCH_LAMB_MOM_TRUST_RATIO_DIV_KERNEL
}

template <typename T, bool NeedUpdate /*=true*/>
struct LambBetaPowUpdateOnceHelper {
  LambBetaPowUpdateOnceHelper(T *beta1pow, T *beta2pow, T beta1, T beta2) {
    PADDLE_ENFORCE_NOT_NULL(beta1pow,
                            platform::errors::InvalidArgument(
                                "The beta1pow should not be nullptr."));
    PADDLE_ENFORCE_NOT_NULL(beta2pow,
                            platform::errors::InvalidArgument(
                                "The beta2pow should not be nullptr."));
    beta1pow_ = beta1pow;
    beta2pow_ = beta2pow;
    beta1_ = beta1;
    beta2_ = beta2;
  }

  HOSTDEVICE void UpdateBetaPows() const {
    beta1pow_[0] *= beta1_;
    beta2pow_[0] *= beta2_;
  }

 private:
  T *__restrict__ beta1pow_;
  T *__restrict__ beta2pow_;
  T beta1_;
  T beta2_;
};

template <typename T>
struct LambBetaPowUpdateOnceHelper<T, false> {
  LambBetaPowUpdateOnceHelper(T *beta1pow, T *beta2pow, T beta1, T beta2) {
    PADDLE_ENFORCE_EQ(
        beta1pow, nullptr,
        platform::errors::InvalidArgument("The beta1pow should be nullptr."));
    PADDLE_ENFORCE_EQ(
        beta2pow, nullptr,
        platform::errors::InvalidArgument("The beta2pow should be nullptr."));
  }

  HOSTDEVICE void UpdateBetaPows() const {}
};

template <typename T, bool HasMasterParam /*=true*/>
struct LambParamHelper {
  LambParamHelper(T *param, MasterT<T> *master_param) {
    constexpr bool kIsSameType = std::is_same<T, MasterT<T>>::value;
    PADDLE_ENFORCE_EQ(kIsSameType, false,
                      platform::errors::InvalidArgument(
                          "T must not be the same with MasterT<T>."));
    PADDLE_ENFORCE_NOT_NULL(master_param,
                            platform::errors::InvalidArgument(
                                "Master parameter must be provided."));
    param_ = param;
    master_param_ = master_param;
  }

  HOSTDEVICE T *__restrict__ ParamPtr() { return param_; }

  HOSTDEVICE MasterT<T> *__restrict__ MasterParamPtr() { return master_param_; }

 private:
  T *__restrict__ param_;
  MasterT<T> *__restrict__ master_param_;
};

template <typename T>
struct LambParamHelper<T, false> {
  LambParamHelper(T *param, MasterT<T> *master_param) {
    constexpr bool kIsSameType = std::is_same<T, MasterT<T>>::value;
    PADDLE_ENFORCE_EQ(kIsSameType, true,
                      platform::errors::InvalidArgument(
                          "T must be the same with MasterT<T>."));
    if (master_param != nullptr) {
      PADDLE_ENFORCE_EQ(static_cast<void *>(param),
                        static_cast<void *>(master_param),
                        platform::errors::InvalidArgument(
                            "Master parameter must be nullptr or the same as "
                            "non-master parameter."));
    }
    param_ = param;
  }

  HOSTDEVICE T *__restrict__ ParamPtr() { return param_; }

  HOSTDEVICE constexpr MasterT<T> *MasterParamPtr() { return nullptr; }

 private:
  T *__restrict__ param_;
};

template <typename ParamT, bool HasMasterParam, bool NeedUpdateBetaPow,
          int VecSize>
struct LambUpdateParamAndBetaPowsFunctor {
  DEVICE void operator()(
      int tensor_id, int chunk_id, int offset, int size,
      LambParamHelper<ParamT, HasMasterParam> param_helper,
      const MasterT<ParamT> *trust_ratio_div, const MasterT<ParamT> *lr,
      const MasterT<ParamT> *param_square_norm,
      const MasterT<ParamT> *trust_ratio_div_square_norm, const bool *found_inf,
      LambBetaPowUpdateOnceHelper<MasterT<ParamT>, NeedUpdateBetaPow>
          betapow_helper) const {
    if (*found_inf) return;

    using MT = MasterT<ParamT>;

    MT p_square_norm = param_square_norm[tensor_id];
    MT t_square_norm = trust_ratio_div_square_norm[tensor_id];
    MT lr_value = *lr;
    MT ratio = (p_square_norm != static_cast<MT>(0) &&
                        t_square_norm != static_cast<MT>(0)
                    ? lr_value * sqrtf(p_square_norm / t_square_norm)
                    : lr_value);

    int i;
    int stride = blockDim.x * VecSize;

    ParamT *param = param_helper.ParamPtr() + offset;
    MT *master_param = HasMasterParam ? param_helper.MasterParamPtr() + offset
                                      : param_helper.MasterParamPtr();
    trust_ratio_div += offset;

    for (i = threadIdx.x * VecSize; i + VecSize <= size; i += stride) {
      platform::AlignedVector<MT, VecSize> trust_ratio_div_vec;
      platform::Load(trust_ratio_div + i, &trust_ratio_div_vec);
      if (HasMasterParam) {
        platform::AlignedVector<MT, VecSize> master_param_vec;
        platform::Load(master_param + i, &master_param_vec);
        platform::AlignedVector<ParamT, VecSize> param_vec;
#pragma unroll
        for (int j = 0; j < VecSize; ++j) {
          MT p = master_param_vec[j] - ratio * trust_ratio_div_vec[j];
          master_param_vec[j] = p;
          param_vec[j] = static_cast<ParamT>(p);
        }
        platform::Store(master_param_vec, master_param + i);
        platform::Store(param_vec, param + i);
      } else {
        platform::AlignedVector<ParamT, VecSize> param_vec;
        platform::Load(param + i, &param_vec);
#pragma unroll
        for (int j = 0; j < VecSize; ++j) {
          MT p = static_cast<MT>(param_vec[j]) - ratio * trust_ratio_div_vec[j];
          param_vec[j] = static_cast<ParamT>(p);
        }
        platform::Store(param_vec, param + i);
      }
    }

    for (; i < size; ++i) {
      if (HasMasterParam) {
        MT p = master_param[i] - ratio * trust_ratio_div[i];
        master_param[i] = p;
        param[i] = static_cast<ParamT>(p);
      } else {
        MT p = static_cast<MT>(param[i]) - ratio * trust_ratio_div[i];
        param[i] = static_cast<ParamT>(p);
      }
    }

    if (NeedUpdateBetaPow && threadIdx.x == 0 && blockIdx.x == 0) {
      betapow_helper.UpdateBetaPows();
    }
  }
};

// TODO(zengjinle): which block_dim and chunk_size would be better?
template <typename ParamT, int MaxTensorNumPerLaunch = 160,
          int MaxChunkNumPerLaunch = 780>
static void MultiTensorUpdateLambParamAndBetaPows(
    const platform::CUDADeviceContext &dev_ctx, const int *offsets, int n,
    const MasterT<ParamT> *trust_ratio_div, const MasterT<ParamT> *lr,
    const MasterT<ParamT> *param_square_norm,
    const MasterT<ParamT> *trust_ratio_div_square_norm, const bool *found_inf,
    ParamT *param, MasterT<ParamT> *master_param, MasterT<ParamT> *beta1pow,
    MasterT<ParamT> *beta2pow, MasterT<ParamT> beta1, MasterT<ParamT> beta2,
    int chunk_size = 65536) {
  constexpr bool kHasMasterParam =
      !(std::is_same<ParamT, MasterT<ParamT>>::value);

  bool has_beta_pow = (beta1pow != nullptr);
  if (has_beta_pow) {
    PADDLE_ENFORCE_NOT_NULL(beta2pow, platform::errors::InvalidArgument(
                                          "Beta2Pow should not be nullptr."));
  } else {
    PADDLE_ENFORCE_EQ(beta2pow, nullptr, platform::errors::InvalidArgument(
                                             "Beta2Pow should be nullptr."));
  }

  const int block_dim = 512;

  int vec_size = 8;
  for (int i = 0; i < n; ++i) {
    int offset = offsets[i] - offsets[0];
    vec_size =
        std::min(vec_size, GetChunkedVecSize(param + offset, chunk_size));
    if (kHasMasterParam) {
      vec_size = std::min(vec_size,
                          GetChunkedVecSize(master_param + offset, chunk_size));
    }
    vec_size = std::min(
        vec_size, GetChunkedVecSize(trust_ratio_div + offset, chunk_size));
  }

  VLOG(1) << __func__ << " VecSize = " << vec_size;

  constexpr auto kNumTensor = MaxTensorNumPerLaunch;
  constexpr auto kNumChunk = MaxChunkNumPerLaunch;

  auto stream = dev_ctx.stream();
#define PD_LAUNCH_MULTI_TENSOR_UPDATE_PARAM_BETAPOW(__has_beta_pow)            \
  do {                                                                         \
    using FunctorT =                                                           \
        LambUpdateParamAndBetaPowsFunctor<ParamT, kHasMasterParam,             \
                                          __has_beta_pow, kVecSize>;           \
    LambParamHelper<ParamT, kHasMasterParam> param_helper(param,               \
                                                          master_param);       \
    LambBetaPowUpdateOnceHelper<MasterT<ParamT>, __has_beta_pow>               \
        betapow_helper(beta1pow, beta2pow, beta1, beta2);                      \
    launcher.Launch(FunctorT(), param_helper, trust_ratio_div, lr,             \
                    param_square_norm, trust_ratio_div_square_norm, found_inf, \
                    betapow_helper);                                           \
  } while (0)

#define PD_LAUNCH_VEC_MULTI_TENSOR_UPDATE_PARAM_BETAPOW_CASE        \
  do {                                                              \
    auto callback = [&](                                            \
        const MultiTensorLauncher<kNumTensor, kNumChunk> &launcher, \
        int launch_n) {                                             \
      if (has_beta_pow && launch_n == 0) {                          \
        PD_LAUNCH_MULTI_TENSOR_UPDATE_PARAM_BETAPOW(true);          \
        beta1pow = nullptr;                                         \
        beta2pow = nullptr;                                         \
      } else {                                                      \
        PD_LAUNCH_MULTI_TENSOR_UPDATE_PARAM_BETAPOW(false);         \
      }                                                             \
    };                                                              \
    MultiTensorApplyWithCallback<kNumTensor, kNumChunk>(            \
        stream, offsets, n, chunk_size, block_dim, callback);       \
  } while (0)

  PD_VEC_LAUNCH_KERNEL(vec_size,
                       PD_LAUNCH_VEC_MULTI_TENSOR_UPDATE_PARAM_BETAPOW_CASE);

#undef PD_LAUNCH_MULTI_TENSOR_UPDATE_PARAM_BETAPOW
#undef PD_LAUNCH_VEC_MULTI_TENSOR_UPDATE_PARAM_BETAPOW_CASE
}

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
static bool CreatePreMulScaleOpIfSupported(ncclDataType_t dtype,
                                           ncclComm_t comm, const void *scale,
                                           ncclRedOp_t *op) {
#if NCCL_VERSION_CODE >= 21100
  int ver;
  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclGetVersion(&ver));
  if (ver >= 21100) {
    VLOG(10) << "ncclRedOpCreatePreMulSum is supported.";
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclRedOpCreatePreMulSum(
        op, const_cast<void *>(scale), dtype, ncclScalarDevice, comm));
    return true;
  }
#endif
  VLOG(10) << "ncclRedOpCreatePreMulSum is not supported.";
  return false;
}

template <typename T1, typename T2>
static void LaunchScaleKernel(const platform::CUDADeviceContext &dev_ctx,
                              const T1 *x, const T2 *scale, T1 *y, int n,
                              gpuStream_t stream) {
  int vec_size = std::min(GetChunkedVecSize(x, 0), GetChunkedVecSize(y, 0));
  auto config = platform::GetGpuLaunchConfig1D(dev_ctx, n, vec_size);

#define PD_LAMB_VEC_SCALE_KERNEL_CASE                                          \
  do {                                                                         \
    ScaleCUDAKernel<T1, T2, kVecSize><<<config.block_per_grid,                 \
                                        config.thread_per_block, 0, stream>>>( \
        x, scale, y, n);                                                       \
  } while (0)

  PD_VEC_LAUNCH_KERNEL(vec_size, PD_LAMB_VEC_SCALE_KERNEL_CASE);
#undef PD_LAMB_VEC_SCALE_KERNEL_CASE
}

template <typename T>
static void NCCLReduceScatterWithScale(
    const T *sendbuff, T *recvbuff, size_t recvcount, size_t nranks,
    ncclComm_t comm, gpuStream_t stream,
    const platform::CUDADeviceContext &dev_ctx, const T *scale = nullptr) {
  static_assert(std::is_same<T, float>::value ||
                    std::is_same<T, platform::float16>::value,
                "T must be either float32 or float16.");
  if (recvcount == 0) return;

  if (comm == nullptr) {
    if (scale != nullptr) {
      PADDLE_ENFORCE_EQ(nranks, 1,
                        platform::errors::InvalidArgument(
                            "nranks must be 1 when scale != nullptr."));
      LaunchScaleKernel(dev_ctx, sendbuff, scale, recvbuff, recvcount * nranks,
                        stream);
    }
    return;
  }

  ncclRedOp_t op = ncclSum;
  ncclDataType_t dtype =
      std::is_same<T, float>::value ? ncclFloat32 : ncclFloat16;
  bool should_destroy_op =
      scale && CreatePreMulScaleOpIfSupported(dtype, comm, scale, &op);
  memory::Buffer buffer(dev_ctx.GetPlace());
  if (scale && !should_destroy_op) {
    size_t numel = recvcount * nranks;
    T *new_sendbuff = buffer.Alloc<T>(numel);
    LaunchScaleKernel(dev_ctx, sendbuff, scale, new_sendbuff, numel, stream);
    sendbuff = new_sendbuff;
  }

  PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclReduceScatter(
      sendbuff, recvbuff, recvcount, dtype, op, comm, stream));

#if NCCL_VERSION_CODE >= 21100
  if (should_destroy_op) {
    VLOG(10) << "ncclRedOpDestroy starts";
    PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclRedOpDestroy(op, comm));
    VLOG(10) << "ncclRedOpDestroy ends";
  }
#endif
}
#endif

template <typename InputIteratorT, typename OutputIteratorT, typename ReduceOpT,
          typename T>
static void CubDeviceReduce(InputIteratorT d_in, OutputIteratorT d_out,
                            int num_items, ReduceOpT reduction_op, T init,
                            gpuStream_t stream, memory::Buffer *buffer) {
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                num_items, reduction_op, init, stream));
  d_temp_storage = buffer->Alloc<void>(temp_storage_bytes);
  VLOG(10) << "hipcub::DeviceReduce::Reduce needs " << temp_storage_bytes
           << " byte(s), ptr = " << d_temp_storage;
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                num_items, reduction_op, init, stream));
}

template <typename T>
static void GetSquareGradNormImpl(const T *grad, int n, float *square_norm,
                                  gpuStream_t stream,
                                  memory::Buffer *cub_tmp_buffer) {
  using Iterator =
      hipcub::TransformInputIterator<float, SquareFunctor<T>, const T *>;
  Iterator iter(grad, SquareFunctor<T>());
  CubDeviceReduce(iter, square_norm, n, hipcub::Sum(), static_cast<float>(0),
                  stream, cub_tmp_buffer);
}

// square_norm is of length 2 at least
static void GetSquareGradNorm(const float *fp32_grad, int fp32_numel,
                              const platform::float16 *fp16_grad,
                              int fp16_numel, float *square_norm,
                              gpuStream_t stream,
                              memory::Buffer *cub_tmp_buffer) {
  VLOG(10) << "GetSquareGradNorm starts, fp32_numel = " << fp32_numel
           << " , fp16_numel = " << fp16_numel;
  if (fp32_numel > 0) {
    GetSquareGradNormImpl(fp32_grad, fp32_numel, square_norm, stream,
                          cub_tmp_buffer);
    VLOG(10) << "FP32 square L2-Norm: "
             << FlattenToString(square_norm, 1, cub_tmp_buffer->GetPlace());
  }

  if (fp16_numel > 0) {
    float *fp16_square_norm = fp32_numel > 0 ? square_norm + 1 : square_norm;
    GetSquareGradNormImpl(fp16_grad, fp16_numel, fp16_square_norm, stream,
                          cub_tmp_buffer);
    VLOG(10) << "FP16 square L2-Norm: "
             << FlattenToString(fp16_square_norm, 1,
                                cub_tmp_buffer->GetPlace());
    if (fp32_numel > 0) {
      AddToCUDAKernel<<<1, 1, 0, stream>>>(fp16_square_norm, square_norm);
      VLOG(10) << "FP32+FP16 square L2-Norm: "
               << FlattenToString(square_norm, 1, cub_tmp_buffer->GetPlace());
    }
  }
  VLOG(10) << "GetSquareGradNorm ends, fp32_numel = " << fp32_numel
           << " , fp16_numel = " << fp16_numel;
}

template <typename T>
std::string NumToString(T x) {
  std::stringstream ss;
  ss << x;
  return ss.str();
}

template <typename T>
static std::string GetMinMaxStr(const T *x, size_t n,
                                const platform::Place &place) {
  PADDLE_ENFORCE_EQ(
      platform::is_gpu_place(place), true,
      platform::errors::InvalidArgument("Only support CUDAPlace currently."));

  auto *dev_ctx = static_cast<platform::CUDADeviceContext *>(
      platform::DeviceContextPool::Instance().Get(place));
  auto stream = dev_ctx->stream();

  memory::Buffer ret_buffer(place);
  T *ret = ret_buffer.Alloc<T>(2);

  if (n > 0) {
    memory::Buffer cub_buffer(place);
    CubDeviceReduce(x, ret, n, hipcub::Min(), std::numeric_limits<T>::max(),
                    stream, &cub_buffer);
    CubDeviceReduce(x, ret + 1, n, hipcub::Max(), std::numeric_limits<T>::lowest(),
                    stream, &cub_buffer);
    T ret_cpu[2];
#ifdef PADDLE_WITH_HIP
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&ret_cpu[0], ret, 2 * sizeof(T),
                                              hipMemcpyDeviceToHost, stream));
    PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&ret_cpu[0], ret, 2 * sizeof(T),
                                               hipMemcpyDeviceToHost, stream));
    PADDLE_ENFORCE_GPU_SUCCESS(hipStreamSynchronize(stream));
#endif
    return std::string("{\"min\": ") + NumToString(ret_cpu[0]) +
           " , \"max\": " + NumToString(ret_cpu[1]) + "}";
  } else {
    return "{\"min\": null, \"max\": null}";
  }
}

struct VisitDTypeFunctor {
  VisitDTypeFunctor(const framework::Tensor *x, std::string *s)
      : x_(x), s_(s) {}

  template <typename T>
  void apply() const {
    *s_ = GetMinMaxStr<T>(x_->template data<T>(), x_->numel(), x_->place());
  }

 private:
  const framework::Tensor *x_;
  std::string *s_;
};

static std::string GetMinMaxStr(const framework::Tensor *x) {
  if (x == nullptr) return "null";
  if (!x->IsInitialized()) return "not_inited";
  if (!platform::is_gpu_place(x->place())) return "CPUTensor";
  std::string str;
  VisitDTypeFunctor functor(x, &str);
  phi::VisitDataType(x->dtype(), functor);
  return str;
}

static void PrintAllMinMaxRange(const framework::ExecutionContext &ctx,
                                bool only_inputs) {
  if (!VLOG_IS_ON(1)) return;
  for (const auto &pair : ctx.GetOp().Inputs()) {
    const auto &key = pair.first;
    const auto tensors = ctx.MultiInput<framework::Tensor>(key);
    size_t n = tensors.size();
    for (size_t i = 0; i < n; ++i) {
      VLOG(1) << "Input(" << key + ")[" << i << "] = " << pair.second[i]
              << " , " << GetMinMaxStr(tensors[i]);
    }
  }

  if (only_inputs) return;
  for (const auto &pair : ctx.GetOp().Outputs()) {
    const auto &key = pair.first;
    const auto tensors = ctx.MultiOutput<framework::Tensor>(key);
    size_t n = tensors.size();
    for (size_t i = 0; i < n; ++i) {
      VLOG(1) << "Output(" << key + ")[" << i << "] = " << pair.second[i]
              << " , " << GetMinMaxStr(tensors[i]);
    }
  }
}

static void CheckHasNanInfGrad(const float *fp32_grad, int fp32_numel,
                               const platform::float16 *fp16_grad,
                               int fp16_numel, float *nan_inf_flag,
                               gpuStream_t stream,
                               memory::Buffer *cub_tmp_buffer) {
  bool *fp32_has_nan_inf = nullptr;
  bool *fp16_has_nan_inf = nullptr;
  if (fp32_numel > 0) {
    fp32_has_nan_inf = reinterpret_cast<bool *>(nan_inf_flag + 1);
    hipcub::TransformInputIterator<bool, IsNanInfFunctor<float>, const float *>
    iter(fp32_grad, IsNanInfFunctor<float>());
    CubDeviceReduce(iter, fp32_has_nan_inf, fp32_numel, OrFunctor(), false,
                    stream, cub_tmp_buffer);
  }

  if (fp16_numel > 0) {
    fp16_has_nan_inf = reinterpret_cast<bool *>(nan_inf_flag + 1) + 1;
    hipcub::TransformInputIterator<bool, IsNanInfFunctor<platform::float16>,
                                const platform::float16 *>
        iter(fp16_grad, IsNanInfFunctor<platform::float16>());
    CubDeviceReduce(iter, fp16_has_nan_inf, fp16_numel, OrFunctor(), false,
                    stream, cub_tmp_buffer);
  }

  if (fp32_has_nan_inf && fp16_has_nan_inf) {
    SetNanInfValueCUDAKernelTwoFlag<<<1, 1, 0, stream>>>(
        fp32_has_nan_inf, fp16_has_nan_inf, nan_inf_flag);
  } else if (fp32_has_nan_inf) {
    SetNanInfValueCUDAKernelOneFlag<<<1, 1, 0, stream>>>(fp32_has_nan_inf,
                                                         nan_inf_flag);
  } else {
    SetNanInfValueCUDAKernelOneFlag<<<1, 1, 0, stream>>>(fp16_has_nan_inf,
                                                         nan_inf_flag);
  }
}

template <typename T>
class DistributedFusedLambOpKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto stream = dev_ctx.stream();
    auto place = dev_ctx.GetPlace();

    // Step 1: Get fp16 param and grad tensors
    int64_t fp16_numel;
    auto *fp16_param = GetSameInOutTensorPtr<platform::float16, true>(
        ctx, place, "FP16FusedParam", "FP16FusedParamOut", &fp16_numel);
    bool has_fp16_param = (fp16_numel > 0);
    const platform::float16 *fp16_grad = nullptr;
    if (has_fp16_param) {
      fp16_grad = GetInputTensorPtr<platform::float16>(ctx, "FP16FusedGrad");
    } else {
      fp16_param = nullptr;
    }

    // Step 2: Get fp32 param and grad tensors
    int64_t fp32_numel = 0;
    auto *fp32_param = GetSameInOutTensorPtr<float, true>(
        ctx, place, "FP32FusedParam", "FP32FusedParamOut", &fp32_numel);
    PADDLE_ENFORCE_GE(fp32_numel, fp16_numel,
                      platform::errors::InvalidArgument(
                          "The element number in FP32FusedParam should be not "
                          "less than FP16FusedParam."));

    fp32_numel -= fp16_numel;  // the FP32FusedParam contains fp32 param and
                               // fp16 master weight
    bool has_fp32_param = (fp32_numel > 0);
    const float *fp32_grad = nullptr;
    if (has_fp32_param) {
      fp32_grad = GetInputTensorPtr<float>(ctx, "FP32FusedGrad");
    } else {
      PADDLE_ENFORCE_EQ(
          has_fp16_param, true,
          platform::errors::InvalidArgument(
              "Either FP32FusedGrad or FP16FusedGrad cannot be NULL."));
    }

    auto numel = fp32_numel + fp16_numel;
    VLOG(1) << "numel = " << numel << " , fp32_numel = " << fp32_numel
            << " , fp16_numel = " << fp16_numel;

    // The NVIDIA cub library does not support number > INT32_MAX
    PADDLE_ENFORCE_LE(numel, std::numeric_limits<int>::max(),
                      platform::errors::Unimplemented(
                          "Too many parameter number. Only <= %d is supported.",
                          std::numeric_limits<int>::max()));

    // Step 3: Get ParamInfo
    const auto *param_info_tensor = GetInputTensorPtr<int>(ctx, "ParamInfo");
    auto fp32_local_start_idx = param_info_tensor[0];
    auto fp32_local_param_num = param_info_tensor[1];
    auto fp32_global_param_num = param_info_tensor[2];
    auto fp32_weight_decay_end_idx = param_info_tensor[3];
    auto fp16_local_start_idx = param_info_tensor[4];
    auto fp16_local_param_num = param_info_tensor[5];
    auto fp16_global_param_num = param_info_tensor[6];
    auto fp16_weight_decay_end_idx = param_info_tensor[7];

    auto local_param_num = fp32_local_param_num + fp16_local_param_num;
    auto param_num = fp32_global_param_num + fp16_global_param_num;
    PADDLE_ENFORCE_LE(local_param_num, param_num,
                      platform::errors::InvalidArgument(
                          "The local parameter number should not exceed the "
                          "global parameter number."));
    VLOG(1) << "local_param_num = " << local_param_num
            << " , global_param_num = " << param_num
            << " , fp32_local_start_idx = " << fp32_local_start_idx
            << " , fp32_local_param_num = " << fp32_local_param_num
            << " , fp32_global_param_num = " << fp32_global_param_num
            << " , fp16_local_start_idx = " << fp16_local_start_idx
            << " , fp16_local_param_num = " << fp16_local_param_num
            << " , fp16_global_param_num = " << fp16_global_param_num;

    // Step 4: Get LearningRate, Moment1, Moment2, Beta1Pow, Beta2Pow,
    // GlobalScale, FoundInf
    const auto *global_scale = GetInputTensorPtr<float>(ctx, "GlobalScale");
    const auto *lr = GetInputTensorPtr<float>(ctx, "LearningRate");
    int64_t partial_numel = 0;
    auto *moment1 = GetSameInOutTensorPtr<float>(ctx, place, "Moment1",
                                                 "Moment1Out", &partial_numel);

    PADDLE_ENFORCE_EQ(numel % partial_numel, 0,
                      platform::errors::InvalidArgument(
                          "The total parameter number %d should be divided "
                          "exactly by the element number %d of Moment1.",
                          numel, partial_numel));

    int64_t num_devices = numel / partial_numel;
    VLOG(1) << "num_devices = " << num_devices
            << " , partial_numel = " << partial_numel;

    PADDLE_ENFORCE_EQ(fp32_numel % num_devices, 0,
                      platform::errors::InvalidArgument(
                          "The fp32 parameter number %d should be divided "
                          "exactly by the device number %d.",
                          fp32_numel, num_devices));
    PADDLE_ENFORCE_EQ(fp16_numel % num_devices, 0,
                      platform::errors::InvalidArgument(
                          "The fp16 parameter number %d should be divided "
                          "exactly by the device number %d.",
                          fp16_numel, num_devices));

    auto *moment2 =
        GetSameInOutTensorPtr<float>(ctx, place, "Moment2", "Moment2Out");
    auto *beta1pow =
        GetSameInOutTensorPtr<float>(ctx, place, "Beta1Pow", "Beta1PowOut");
    auto *beta2pow =
        GetSameInOutTensorPtr<float>(ctx, place, "Beta2Pow", "Beta2PowOut");

    auto *found_inf_t = ctx.Output<framework::Tensor>("FoundInf");
    found_inf_t->Resize({1});
    auto *found_inf = found_inf_t->mutable_data<bool>(place);

    // Step 5: Get attributes weight_decay, beta1, beta2, epsilon,
    // max_grad_norm, ring_id,
    // use_master_param_norm, is_grad_scaled_by_nranks
    auto weight_decay = ctx.Attr<float>("weight_decay");
    auto beta1 = ctx.Attr<float>("beta1");
    auto beta2 = ctx.Attr<float>("beta2");
    auto epsilon = ctx.Attr<float>("epsilon");
    auto max_global_grad_norm = ctx.Attr<float>("max_global_grad_norm");
    auto clip_after_allreduce = ctx.Attr<bool>("clip_after_allreduce");
    auto ring_id = ctx.Attr<int>("ring_id");
    auto use_master_param_norm = ctx.Attr<bool>("use_master_param_norm");
    auto is_grad_scaled_by_nranks = ctx.Attr<bool>("is_grad_scaled_by_nranks");
    VLOG(10) << "max_global_grad_norm = " << max_global_grad_norm
             << " , clip_after_allreduce = " << clip_after_allreduce
             << " , use_master_param_norm = " << use_master_param_norm
             << " , is_grad_scaled_by_nranks = " << is_grad_scaled_by_nranks;

    // Step 6: allreduce + global norm gradient clip
    int rank = 0;
    ncclComm_t comm = nullptr;
    if (num_devices > 1) {
      auto *nccl_comm_handle =
          platform::NCCLCommContext::Instance().Get(ring_id, place);
      comm = nccl_comm_handle->comm();
      rank = nccl_comm_handle->rank();
    }

    memory::Buffer grad_norm_square_buffer(place);
    auto *fp32_square_grad_norm = grad_norm_square_buffer.Alloc<float>(2);
    memory::Buffer cub_tmp_buffer(place);

    memory::Buffer sum_grad_buffer(place);
    float *fp32_sum_grad;
    platform::float16 *fp16_sum_grad;
    auto fp32_numel_each_device = fp32_numel / num_devices;
    auto fp16_numel_each_device = fp16_numel / num_devices;
    if (num_devices > 1 ||
        (max_global_grad_norm > 0 && !clip_after_allreduce)) {
      auto ptr = sum_grad_buffer.Alloc<uint8_t>(
          fp32_numel_each_device * sizeof(float) +
          fp16_numel_each_device * sizeof(platform::float16));
      fp32_sum_grad = has_fp32_param ? reinterpret_cast<float *>(ptr) : nullptr;
      fp16_sum_grad = has_fp16_param
                          ? reinterpret_cast<platform::float16 *>(
                                ptr + fp32_numel_each_device * sizeof(float))
                          : nullptr;
    } else {
      // NOTE: The const_cast here is not important. The fp32_sum_grad and
      // fp16_sum_grad would not be changed when num_devices == 1
      // But if I do not perform const_cast here, there would be more
      // if-else codes (num_devices > 1) when I write the following code.
      // So I prefer to use const_cast to unify the following code to reduce
      // the if-else codes.
      fp32_sum_grad = const_cast<float *>(fp32_grad);
      fp16_sum_grad = const_cast<platform::float16 *>(fp16_grad);
    }

    float rescale_grad = 1.0f;
    if (!is_grad_scaled_by_nranks) {
      rescale_grad /= num_devices;
    }

    if (max_global_grad_norm > 0) {
      if (clip_after_allreduce) {
        // (1) ReduceScater first
        NCCLReduceScatterWithScale(fp32_grad, fp32_sum_grad,
                                   fp32_numel_each_device, num_devices, comm,
                                   stream, dev_ctx);
        NCCLReduceScatterWithScale(fp16_grad, fp16_sum_grad,
                                   fp16_numel_each_device, num_devices, comm,
                                   stream, dev_ctx);
        // (2) Calculate the global grad norm
        GetSquareGradNorm(fp32_sum_grad, fp32_numel_each_device, fp16_sum_grad,
                          fp16_numel_each_device, fp32_square_grad_norm, stream,
                          &cub_tmp_buffer);
        VLOG(1) << "Grad square norm before all reduce: "
                << FlattenToString(fp32_square_grad_norm, 1, place);
        if (num_devices > 1) {
          PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
              fp32_square_grad_norm, fp32_square_grad_norm, 1, ncclFloat32,
              ncclSum, comm, stream));
        }
        VLOG(1) << "Grad square norm after all reduce: "
                << FlattenToString(fp32_square_grad_norm, 1, place);
      } else {
        // (1) Calculate the local grad norm
        GetSquareGradNorm(fp32_grad, fp32_numel, fp16_grad, fp16_numel,
                          fp32_square_grad_norm, stream, &cub_tmp_buffer);
        VLOG(1) << "Grad square norm before all reduce: "
                << FlattenToString(fp32_square_grad_norm, 1, place);
        // (2) Calculate the gradient clip scale
        float *fp32_scale = nullptr;
        platform::float16 *fp16_scale = nullptr;
        if (has_fp32_param && has_fp16_param) {
          auto *ptr = cub_tmp_buffer.Alloc<uint8_t>(sizeof(float) +
                                                    sizeof(platform::float16));
          fp32_scale = reinterpret_cast<float *>(ptr);
          fp16_scale =
              reinterpret_cast<platform::float16 *>(ptr + sizeof(float));
        } else if (has_fp32_param) {
          fp32_scale = cub_tmp_buffer.Alloc<float>(1);
        } else {
          fp16_scale = cub_tmp_buffer.Alloc<platform::float16>(1);
        }

        float clip_scale = 1.0f;
        if (is_grad_scaled_by_nranks) {
          clip_scale *= num_devices;
        }
        CalcGradNormClipBeforeAllReduceScale<
            float, platform::float16><<<1, 1, 0, stream>>>(
            global_scale, max_global_grad_norm, fp32_square_grad_norm,
            fp32_scale, fp16_scale, clip_scale);
        if (fp32_scale) {
          VLOG(1) << "Grad scale: " << FlattenToString(fp32_scale, 1, place);
        } else {
          VLOG(1) << "Grad scale: " << FlattenToString(fp16_scale, 1, place);
        }
        if (num_devices > 1) {
          PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
              fp32_square_grad_norm, fp32_square_grad_norm, 1, ncclFloat32,
              ncclSum, comm, stream));
        }
        // (3) Do ReduceScatter with scale
        NCCLReduceScatterWithScale(fp32_grad, fp32_sum_grad,
                                   fp32_numel_each_device, num_devices, comm,
                                   stream, dev_ctx, fp32_scale);
        NCCLReduceScatterWithScale(fp16_grad, fp16_sum_grad,
                                   fp16_numel_each_device, num_devices, comm,
                                   stream, dev_ctx, fp16_scale);
        // (4) mark max_global_grad_norm as 0, meaning that clip has been
        // already performed
        max_global_grad_norm = 0;
      }
    } else {
      NCCLReduceScatterWithScale(fp32_grad, fp32_sum_grad,
                                 fp32_numel_each_device, num_devices, comm,
                                 stream, dev_ctx);
      NCCLReduceScatterWithScale(fp16_grad, fp16_sum_grad,
                                 fp16_numel_each_device, num_devices, comm,
                                 stream, dev_ctx);
      CheckHasNanInfGrad(fp32_sum_grad, fp32_numel_each_device, fp16_sum_grad,
                         fp16_numel_each_device, fp32_square_grad_norm, stream,
                         &cub_tmp_buffer);
      if (num_devices > 1) {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
            fp32_square_grad_norm, fp32_square_grad_norm, 1, ncclFloat32,
            ncclSum, comm, stream));
      }
      max_global_grad_norm = 0;
    }
    VLOG(10) << "ReduceScatter done";

    // Step 7: update the moment1, moment2. Calcuate the trust_ratio_div
    auto *fused_offsets_t = ctx.Input<framework::Tensor>("FusedParamOffsets");
    auto *fused_offsets = fused_offsets_t->data<int>();
    auto *fp32_partial_fused_offsets_t =
        ctx.Input<framework::Tensor>("FP32ShardFusedParamOffsets");
    const auto *fp32_partial_fused_offsets =
        fp32_partial_fused_offsets_t->data<int>();
    auto *fp16_partial_fused_offsets_t =
        ctx.Input<framework::Tensor>("FP16ShardFusedParamOffsets");
    const auto *fp16_partial_fused_offsets =
        fp16_partial_fused_offsets_t->data<int>();

    VLOG(1) << "FusedParamOffsets: "
            << FlattenToString(fused_offsets, fused_offsets_t->numel(),
                               fused_offsets_t->place());
    VLOG(1) << "FP32ShardFusedParamOffsets: "
            << FlattenToString(fp32_partial_fused_offsets,
                               fp32_partial_fused_offsets_t->numel(),
                               fp32_partial_fused_offsets_t->place());
    VLOG(1) << "FP16ShardFusedParamOffsets: "
            << FlattenToString(fp16_partial_fused_offsets,
                               fp16_partial_fused_offsets_t->numel(),
                               fp16_partial_fused_offsets_t->place());

    memory::Buffer trust_ratio_div_buffer(place);
    auto *trust_ratio_div = trust_ratio_div_buffer.Alloc<float>(partial_numel);
    auto fp32_offset = rank * fp32_numel_each_device;
    auto fp16_offset = rank * fp16_numel_each_device;
    if (has_fp32_param) {
      VLOG(10) << "Update FP32 Moment and TrustRatioDiv starts";
      MultiTensorUpdateLambMomentAndTrustRatioDiv(
          dev_ctx, fp32_partial_fused_offsets, fp32_local_param_num,
          fp32_param + fp32_offset, fp32_sum_grad, fp32_square_grad_norm,
          global_scale, beta1pow, beta2pow, moment1, moment2, trust_ratio_div,
          found_inf, weight_decay, fp32_weight_decay_end_idx, beta1, beta2,
          epsilon, max_global_grad_norm, rescale_grad);
      VLOG(10) << "Update FP32 Moment and TrustRatioDiv done";
    }
    float *master_param = nullptr;
    if (has_fp16_param) {
      master_param = fp32_param + fp32_numel;
      VLOG(10) << "Update FP16 Moment and TrustRatioDiv starts";
      auto tmp_found_inf = has_fp32_param ? nullptr : found_inf;
      MultiTensorUpdateLambMomentAndTrustRatioDiv(
          dev_ctx, fp16_partial_fused_offsets, fp16_local_param_num,
          master_param + fp16_offset, fp16_sum_grad, fp32_square_grad_norm,
          global_scale, beta1pow, beta2pow, moment1 + fp32_numel_each_device,
          moment2 + fp32_numel_each_device,
          trust_ratio_div + fp32_numel_each_device, tmp_found_inf, weight_decay,
          fp16_weight_decay_end_idx, beta1, beta2, epsilon,
          max_global_grad_norm, rescale_grad);
      VLOG(10) << "Update FP16 Moment and TrustRatioDiv done";
    }

    VLOG(10) << "Update Moment and TrustRatioDiv done hehahaha";

    // Step 8: calculate L2-Norm square of parameter and trust_ratio_div
    memory::Buffer square_norm_buffer(place);
    auto *param_square_norm = square_norm_buffer.Alloc<float>(2 * param_num);
    auto *trust_ratio_div_square_norm = param_square_norm + param_num;
    if (num_devices > 1) {
      if (use_master_param_norm) {
        FillZeroWithPtr(param_square_norm + fp32_global_param_num,
                        2 * param_num - fp32_global_param_num, stream);
      } else {
        FillZeroWithPtr(trust_ratio_div_square_norm, param_num, stream);
      }
    }
    MultiTensorL2Norm(place, stream, fp32_param, fused_offsets,
                      fp32_global_param_num, param_square_norm);
    if (use_master_param_norm) {
      MultiTensorL2Norm(place, stream, master_param + fp16_offset,
                        fp16_partial_fused_offsets, fp16_local_param_num,
                        param_square_norm + fp16_local_start_idx);
    } else {
      MultiTensorL2Norm(
          place, stream, fp16_param + fused_offsets[fp16_local_start_idx] -
                             fused_offsets[fp32_global_param_num],
          fused_offsets + fp16_local_start_idx, fp16_local_param_num,
          param_square_norm + fp16_local_start_idx);
    }

    MultiTensorL2Norm(place, stream, trust_ratio_div,
                      fp32_partial_fused_offsets, fp32_local_param_num,
                      trust_ratio_div_square_norm + fp32_local_start_idx);
    MultiTensorL2Norm(place, stream, trust_ratio_div + fp32_numel_each_device,
                      fp16_partial_fused_offsets, fp16_local_param_num,
                      trust_ratio_div_square_norm + fp16_local_start_idx);

    VLOG(1) << "TrustRatioDiv L2-Norm before allreduce: "
            << FlattenToString(trust_ratio_div_square_norm, param_num, place);
    if (num_devices > 1) {
      if (use_master_param_norm) {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
            param_square_norm + fp32_global_param_num,
            param_square_norm + fp32_global_param_num,
            2 * param_num - fp32_global_param_num, ncclFloat32, ncclSum, comm,
            stream));
      } else {
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllReduce(
            trust_ratio_div_square_norm, trust_ratio_div_square_norm, param_num,
            ncclFloat32, ncclSum, comm, stream));
      }
      VLOG(10) << "ncclAllReduce done";
    }

    LogParamAndTrustRatioDivSquareNorm<1>(ctx, param_square_norm,
                                          trust_ratio_div_square_norm);
    VLOG(10) << "Calculate L2-Norm of Param and TrustRatioDiv done";

    // Step 9: update parameter, beta1pow, beta2pow. All gather parameters.
    if (has_fp32_param) {
      MultiTensorUpdateLambParamAndBetaPows<float>(
          dev_ctx, fp32_partial_fused_offsets, fp32_local_param_num,
          trust_ratio_div, lr, param_square_norm + fp32_local_start_idx,
          trust_ratio_div_square_norm + fp32_local_start_idx, found_inf,
          fp32_param + fp32_offset, nullptr, beta1pow, beta2pow, beta1, beta2);
      if (num_devices > 1) {
        // ncclAllGather
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllGather(
            fp32_param + fp32_offset, fp32_param, fp32_numel_each_device,
            ncclFloat32, comm, stream));
      }

      beta1pow = nullptr;
      beta2pow = nullptr;
    }
    if (has_fp16_param) {
      MultiTensorUpdateLambParamAndBetaPows<platform::float16>(
          dev_ctx, fp16_partial_fused_offsets, fp16_local_param_num,
          trust_ratio_div + fp32_numel_each_device, lr,
          param_square_norm + fp16_local_start_idx,
          trust_ratio_div_square_norm + fp16_local_start_idx, found_inf,
          fp16_param + fp16_offset, master_param + fp16_offset, beta1pow,
          beta2pow, beta1, beta2);
      if (num_devices > 1) {
        // ncclAllGather
        PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::ncclAllGather(
            fp16_param + fp16_offset, fp16_param, fp16_numel_each_device,
            ncclFloat16, comm, stream));
      }
    }
    VLOG(10) << "Update Param done";

    VLOG(1) << "IsFinite: " << IsFinite(dev_ctx, fp32_square_grad_norm);
#else
    PADDLE_THROW(platform::errors::Unimplemented(
        "distributed_fused_lamb op should be used with NCCL/RCCL."));
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace plat = paddle::platform;
namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    distributed_fused_lamb,
    ops::DistributedFusedLambOpKernel<plat::CUDADeviceContext, float>);
