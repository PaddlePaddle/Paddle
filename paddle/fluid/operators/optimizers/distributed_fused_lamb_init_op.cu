#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/operators/optimizers/cast_with_ptr.h"
#include "paddle/fluid/operators/optimizers/distributed_fused_lamb_init_op.h"
#include "paddle/fluid/operators/tensor_to_string.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/kernels/funcs/algorithm.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace paddle {
namespace operators {

struct ParamGradInfo {
  framework::Tensor *param_t{nullptr};
  framework::Tensor *grad_t{nullptr};
  size_t idx{0};
  size_t numel{0};
  size_t numel_with_padding{0};
  size_t numel_offset{0};
};

static std::ostream &operator<<(std::ostream &os, const ParamGradInfo &info) {
  return os << "{Param(" << info.param_t << "),Grad(" << info.grad_t << "),idx("
            << info.idx << "),numel(" << info.numel << "),numel_with_padding("
            << info.numel_with_padding << "),numel_offset(" << info.numel_offset
            << "),padding(" << info.numel_offset + info.numel_with_padding
            << "-" << info.numel_offset + info.numel << "="
            << info.numel_with_padding - info.numel << ")}";
}

struct ParamGradInfoNumelOffsetCompFunctor {
  bool operator()(const ParamGradInfo &x, const ParamGradInfo &y) const {
    return x.numel_offset < y.numel_offset;
  }

  bool operator()(const ParamGradInfo &x, size_t y) const {
    return x.numel_offset < y;
  }

  bool operator()(size_t x, const ParamGradInfo &y) const {
    return x < y.numel_offset;
  }

  bool operator()(size_t x, size_t y) const { return x < y; }
};

static size_t GetAlignSize(size_t n, size_t alignment) {
  auto remainder = n % alignment;
  return remainder == 0 ? n : n + alignment - remainder;
}

// Shard the ParamGradInfo list by the numel size [start_size, end_size)
// The final results should be:
//
// start_size = sum(infos[0:i].numel_with_padding) + start_numel_offset, where
// start_numel_offset <= infos[i].numel_with_padding
//
// end_size = sum(infos[0:j].numel_with_padding) + end_numel_offset, where
// end_numel_offset <= infos[j].numel_with_padding
static void GetParamGradShardInfo(const std::vector<ParamGradInfo> &infos,
                                  size_t start_size, size_t end_size,
                                  size_t *start_idx, size_t *end_idx,
                                  size_t *start_numel_offset,
                                  size_t *end_numel_offset) {
  VLOG(10) << "NumelOffset: "
           << string::join_strings(infos, ",", [](const ParamGradInfo &info) {
                return info.numel_offset;
              });
  VLOG(10) << "start_size = " << start_size << " , end_size = " << end_size;

  if (infos.empty()) {
    PADDLE_ENFORCE_EQ(start_size, 0, platform::errors::InvalidArgument(
                                         "start_size should be 0."));
    PADDLE_ENFORCE_EQ(end_size, 0, platform::errors::InvalidArgument(
                                       "end_size should be 0."));
    *start_idx = 0;
    *end_idx = 0;
    *start_numel_offset = 0;
    *end_numel_offset = 0;
    return;
  }

  PADDLE_ENFORCE_LT(start_size, end_size,
                    platform::errors::InvalidArgument(
                        "start_size should be less than end_size."));
  size_t n = infos.size();
  ParamGradInfoNumelOffsetCompFunctor comp;
  auto i = static_cast<size_t>(
      std::lower_bound(infos.begin(), infos.end(), start_size, comp) -
      infos.begin());
  if (i == n || infos[i].numel_offset != start_size) {
    PADDLE_ENFORCE_GT(
        i, 0, platform::errors::InvalidArgument(
                  "Cannot find suitable sharding which is between [%d, %d)",
                  start_size, end_size));
    --i;
  }
  PADDLE_ENFORCE_LT(
      i, n, platform::errors::InvalidArgument(
                "Cannot find suitable sharding which is between [%d, %d)",
                start_size, end_size));
  *start_idx = i;
  *start_numel_offset = start_size - infos[i].numel_offset;
  auto j = static_cast<size_t>(
      std::lower_bound(infos.begin(), infos.end(), end_size, comp) -
      infos.begin());
  *end_idx = j - 1;
  *end_numel_offset = end_size - infos[j - 1].numel_offset;
  PADDLE_ENFORCE_GT(*end_numel_offset, 0,
                    platform::errors::InvalidArgument(
                        "Internal error when sharding, this may be a bug "
                        "caused by empty parameter."));
  VLOG(10) << "Sharding [start_size=" << start_size << ", end_size=" << end_size
           << "): " << (*start_idx) << ":" << (*start_numel_offset) << " -> "
           << (*end_idx) << ":" << (*end_numel_offset);
}

static size_t FillAlignmentPaddingInfo(std::vector<ParamGradInfo> *infos,
                                       size_t alignment, size_t nranks,
                                       phi::DataType dtype) {
  auto sizeof_dtype = paddle::experimental::SizeOf(dtype);
  PADDLE_ENFORCE_EQ(
      alignment % sizeof_dtype, 0,
      platform::errors::InvalidArgument(
          "The attr(alignment) should be exactly divided by sizeof(T) %d.",
          sizeof_dtype));
  alignment /= sizeof_dtype;

  size_t total_numel_sum_with_padding = 0;
  size_t n = infos->size();
  for (size_t i = 0; i < n; ++i) {
    auto &info = (*infos)[i];
    size_t numel_with_padding;
    if (i + 1 == n) {
      // the total fused numel must be a factor of alignment * nranks
      numel_with_padding =
          GetAlignSize(info.numel + total_numel_sum_with_padding,
                       alignment * nranks) -
          total_numel_sum_with_padding;
    } else {
      numel_with_padding = GetAlignSize(info.numel, alignment);
    }
    info.numel_with_padding = numel_with_padding;
    info.numel_offset = total_numel_sum_with_padding;
    total_numel_sum_with_padding += numel_with_padding;
  }
  return total_numel_sum_with_padding;
}

template <typename T>
static T *TensorFillConstant(const platform::CUDADeviceContext &dev_ctx,
                             framework::Tensor *tensor,
                             const framework::DDim &dims, T value) {
  tensor->Resize(dims);
  auto *ptr = tensor->mutable_data<T>(dev_ctx.GetPlace());
  phi::funcs::SetConstant<platform::CUDADeviceContext, T> set_constant;
  set_constant(dev_ctx, tensor, value);
  return ptr;
}

static framework::Tensor CastDataForInitedTensor(
    const platform::CUDADeviceContext &dev_ctx, framework::Tensor *origin,
    framework::Tensor *fused_out, size_t numel_offset) {
  PADDLE_ENFORCE_EQ(origin->IsInitialized(), true,
                    platform::errors::InvalidArgument(
                        "The tensor to be cast should be initialized."));

  PADDLE_ENFORCE_EQ(fused_out->dtype(), phi::DataType::FLOAT32,
                    platform::errors::InvalidArgument(
                        "The dst tensor to be cast should be FP32 tensor."));
  PADDLE_ENFORCE_EQ(origin->dtype(), phi::DataType::FLOAT16,
                    platform::errors::InvalidArgument(
                        "The src tensor to be cast should be FP16 tensor."));
  auto *dst = fused_out->data<float>() + numel_offset;
  auto *src = origin->data<platform::float16>();
  auto numel = origin->numel();
  LaunchCastKernel(dev_ctx, src, dst, numel);
  VLOG(10) << "Cast from FP32 -> FP16, range: [" << numel_offset << ", "
           << numel_offset + numel << ")"
           << " , total: [0, " << fused_out->numel() << ")";
  framework::DDim fused_out_dim = fused_out->dims();
  auto fused_out_numel = fused_out->numel();
  fused_out->Resize({fused_out_numel});
  auto sliced_tensor = fused_out->Slice(numel_offset, numel + numel_offset);
  fused_out->Resize(fused_out_dim);
  return sliced_tensor;
}

static framework::Tensor CopyAndShareBufferForInitedTensor(
    framework::Tensor *origin, framework::Tensor *fused_out,
    size_t numel_offset, gpuStream_t stream) {
  PADDLE_ENFORCE_EQ(
      origin->IsInitialized(), true,
      platform::errors::InvalidArgument(
          "The tensor to be copied and shared data should be initialized."));
  auto dtype = fused_out->type();
  PADDLE_ENFORCE_EQ(origin->type(), dtype,
                    platform::errors::InvalidArgument(
                        "The tensor to be copied and shared data should be "
                        "have the same data type."));
  auto place = fused_out->place();
  PADDLE_ENFORCE_EQ(
      origin->place(), place,
      platform::errors::InvalidArgument("The tensor to be copied and shared "
                                        "data should be have the same place."));
  PADDLE_ENFORCE_EQ(
      platform::is_gpu_place(place), true,
      platform::errors::InvalidArgument(
          "The tensor to be copied and shared data should be on GPU place."));

  auto numel = origin->numel();
  framework::DDim fused_out_dim = fused_out->dims();
  auto fused_out_numel = fused_out->numel();
  auto sliced_tensor = fused_out->Resize({fused_out_numel})
                           .Slice(numel_offset, numel + numel_offset);
  memory::Copy(place, sliced_tensor.data(), place, origin->data(),
               numel * paddle::experimental::SizeOf(dtype), stream);
  origin->ShareBufferWith(sliced_tensor);
  fused_out->Resize(fused_out_dim);
  VLOG(10) << "Copy and share buffer, range: [" << numel_offset << ", "
           << numel_offset + numel << ") , total: [0, " << fused_out->numel()
           << ") , dtype = " << dtype;
  return sliced_tensor;
}

static void ShareBufferForNonInitedTensor(framework::Tensor *origin,
                                          framework::Tensor *fused_out,
                                          size_t numel_offset,
                                          const framework::DDim &dims) {
  PADDLE_ENFORCE_EQ(
      origin->IsInitialized(), false,
      platform::errors::InvalidArgument(
          "The tensor to be shared data should not be initialized."));

  framework::DDim fused_out_dim = fused_out->dims();
  auto fused_out_numel = fused_out->numel();
  auto numel = phi::product(dims);
  *origin = fused_out->Resize({fused_out_numel})
                .Slice(numel_offset, numel + numel_offset);
  origin->Resize(dims);
  fused_out->Resize(fused_out_dim);
  VLOG(10) << "Share buffer for non-inited, range: [" << numel_offset << ", "
           << numel_offset + numel << "), total: [0, " << fused_out->numel()
           << ") , dtype = " << fused_out->dtype();
}

template <typename T>
static void CopyVectorToCPUTensor(const std::vector<T> &src,
                                  framework::Tensor *dst) {
  dst->Resize({static_cast<int64_t>(src.size())});
  T *dst_ptr = dst->mutable_data<T>(platform::CPUPlace());
  const T *src_ptr = src.data();
  auto nbytes = src.size() * sizeof(T);
  std::memcpy(dst_ptr, src_ptr, nbytes);
}

static size_t ReorderParamGradInfoList(const std::vector<int> &flags,
                                       std::vector<ParamGradInfo> *infos) {
  size_t n = infos->size();
  std::vector<int> cur_flags;
  cur_flags.reserve(n);
  for (size_t i = 0; i < n; ++i) {
    auto idx = (*infos)[i].idx;
    cur_flags.push_back(flags[idx]);
  }

  auto origin_infos = *infos;
  size_t j = 0;
  for (size_t i = 0; i < n; ++i) {
    if (cur_flags[i]) {
      (*infos)[j] = origin_infos[i];
      ++j;
    }
  }
  size_t ret_idx = j;

  for (size_t i = 0; i < n; ++i) {
    if (!cur_flags[i]) {
      (*infos)[j] = origin_infos[i];
      ++j;
    }
  }
  return ret_idx;
}

template <typename T>
static T ClipByBound(T x, T low_value, T high_value) {
  if (x < low_value) return low_value;
  if (x > high_value) return high_value;
  return x;
}

template <typename T>
class DistributedFusedLambInitOpKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    VLOG(10) << "starts to run DistributedFusedLambInitOp";
    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    auto place = ctx.GetPlace();
    auto stream = dev_ctx.stream();

    // Step 1: Check Input(Param) and Output(ParamOut), Input(Grad) and
    // Output(GradOut)
    auto params = ctx.MultiInput<framework::Tensor>("Param");
    auto grads = ctx.MultiInput<framework::Tensor>("Grad");
    auto master_params = ctx.MultiOutput<framework::Tensor>("MasterParamOut");
    std::vector<ParamGradInfo> fp32_infos, fp16_infos;
    {
      PADDLE_ENFORCE_EQ(params.size(), grads.size(),
                        platform::errors::InvalidArgument(
                            "The parameter number and parameter gradient "
                            "number should be the same."));

      auto params_out = ctx.MultiOutput<framework::Tensor>("ParamOut");
      auto grads_out = ctx.MultiOutput<framework::Tensor>("GradOut");
      PADDLE_ENFORCE_EQ(
          params.size(), params_out.size(),
          platform::errors::InvalidArgument("Input(Param) and Output(ParamOut) "
                                            "should have the same number."));
      PADDLE_ENFORCE_EQ(
          grads.size(), grads_out.size(),
          platform::errors::InvalidArgument(
              "Input(Grad) and Output(GradOut) should have the same number."));
      size_t n = params.size();
      VLOG(10) << "parameter number: " << n;
      for (size_t i = 0; i < n; ++i) {
        auto *p = params[i];
        auto *g = grads[i];
        auto *p_out = params_out[i];
        auto *g_out = grads_out[i];

        PADDLE_ENFORCE_NOT_NULL(
            p, platform::errors::InvalidArgument(
                   "The %d-th parameter should not be nullptr.", i));
        PADDLE_ENFORCE_EQ(p->IsInitialized(), true,
                          platform::errors::InvalidArgument(
                              "The %d-th parameter should be initialized.", i));
        PADDLE_ENFORCE_EQ(
            p->place(), place,
            platform::errors::InvalidArgument(
                "The %d-th parameter is not initialized on the right place.",
                i));
        PADDLE_ENFORCE_EQ(p, p_out,
                          platform::errors::InvalidArgument(
                              "The %d-th Input(Param) and Output(ParamOut) "
                              "should be the same tensor.",
                              i));

        auto dtype = p->dtype();
        PADDLE_ENFORCE_NOT_NULL(
            g, platform::errors::InvalidArgument(
                   "The %d-th gradient should not be nullptr.", i));
        PADDLE_ENFORCE_EQ(g, g_out,
                          platform::errors::InvalidArgument(
                              "The %d-th Input(Grad) and Output(Grad) should "
                              "be the same tensor."));
        auto numel = p->numel();
        PADDLE_ENFORCE_GT(numel, 0,
                          platform::errors::InvalidArgument(
                              "The %d-th Input(Param) have no elements."));

        void *g_data = nullptr;
        if (g->IsInitialized()) {
          PADDLE_ENFORCE_EQ(g->dtype(), dtype,
                            platform::errors::InvalidArgument(
                                "The %d-th Input(Param) and Input(Grad) should "
                                "have the same data type %s.",
                                i, dtype));
          PADDLE_ENFORCE_EQ(g->dims(), p->dims(),
                            platform::errors::InvalidArgument(
                                "The %d-th Input(Param) and Input(Grad) should "
                                "have the same shape.",
                                i));
          g_data = g_out->data();
        }

        ParamGradInfo *info;
        if (dtype == phi::DataType::FLOAT32) {
          fp32_infos.emplace_back();
          info = &fp32_infos.back();
        } else if (dtype == phi::DataType::FLOAT16) {
          fp16_infos.emplace_back();
          info = &fp16_infos.back();
        } else {
          PADDLE_THROW(platform::errors::InvalidArgument(
              "Unsupported data type %s.", dtype));
        }

        VLOG(10) << "Found " << dtype << " parameter " << i << " shape=["
                 << p_out->dims() << "] numel=" << numel
                 << " grad.IsInitialized()="
                 << (g_out->IsInitialized() ? "true" : "false");

        info->param_t = p_out;
        info->grad_t = g_out;
        info->idx = i;
        info->numel = numel;
        info->numel_with_padding = 0;  // not determined yet
        info->numel_offset = 0;        // not determined yet
      }
    }
    const auto &apply_weight_decay =
        ctx.Attr<std::vector<int>>("apply_weight_decay");
    size_t fp32_wd_end_idx =
        ReorderParamGradInfoList(apply_weight_decay, &fp32_infos);
    size_t fp16_wd_end_idx =
        ReorderParamGradInfoList(apply_weight_decay, &fp16_infos);

    auto *param_order_t = ctx.Output<framework::Tensor>("ParamOrder");
    auto param_num = fp32_infos.size() + fp16_infos.size();
    param_order_t->Resize({static_cast<int16_t>(param_num)});
    auto *param_order = param_order_t->mutable_data<int>(platform::CPUPlace());
    for (size_t i = 0; i < fp32_infos.size(); ++i) {
      param_order[i] = static_cast<int>(fp32_infos[i].idx);
    }
    for (size_t i = 0; i < fp16_infos.size(); ++i) {
      param_order[i + fp32_infos.size()] = static_cast<int>(fp16_infos[i].idx);
    }

    VLOG(10) << "Fill ParamGradInfo ends";

    // Step 2: determine the numel_with_padding and numel_offset
    auto rank = ctx.Attr<int>("rank");
    auto nranks = ctx.Attr<int>("nranks");
    auto alignment = ctx.Attr<int>("alignment");
    VLOG(10) << "rank = " << rank << ", nranks = " << nranks
             << " , alignment = " << alignment;
    if (alignment <= 0) {
      alignment = platform::GpuMinChunkSize();
    }
    PADDLE_ENFORCE_GE(alignment, 1,
                      platform::errors::InvalidArgument(
                          "The attr(alignment) should be larger than 0."));
    PADDLE_ENFORCE_EQ(alignment & (alignment - 1), 0,
                      platform::errors::InvalidArgument(
                          "The attr(alignment) should be the power of 2."));
    PADDLE_ENFORCE_GE(
        rank, 0, platform::errors::InvalidArgument(
                     "The attr(rank) should be equal to or larger than 0."));
    PADDLE_ENFORCE_LT(
        rank, nranks,
        platform::errors::InvalidArgument(
            "The attr(rank) should be less than the attr(nranks)."));
    // NOTE: We guarantee that both fp32_numel and fp16_numel can be exactly
    // divided by alignment and nranks.
    auto fp32_numel = FillAlignmentPaddingInfo(&fp32_infos, alignment, nranks,
                                               phi::DataType::FLOAT32);
    VLOG(10) << "FP32 ParamGradInfo: " << string::join_strings(fp32_infos, " ");
    auto fp16_numel = FillAlignmentPaddingInfo(&fp16_infos, alignment, nranks,
                                               phi::DataType::FLOAT16);
    VLOG(10) << "FP16 ParamGradInfo: " << string::join_strings(fp16_infos, " ");
    auto total_numel = fp32_numel + fp16_numel;
    PADDLE_ENFORCE_LT(
        total_numel, std::numeric_limits<int>::max(),
        platform::errors::InvalidArgument("Too many parameter number."));

    auto fp32_numel_each_device = fp32_numel / nranks;
    auto fp16_numel_each_device = fp16_numel / nranks;
    auto numel_each_device = fp32_numel_each_device + fp16_numel_each_device;
    VLOG(10) << "Fill padding ends. total_numel = " << total_numel
             << ", fp32_numel = " << fp32_numel
             << ", fp16_numel = " << fp16_numel
             << ", fp32_numel_each_device = " << fp32_numel_each_device
             << ", fp16_numel_each_device = " << fp16_numel_each_device;

    // Step 3: allocate output tensor and do initialization
    float *fused_fp32_param = nullptr, *fused_fp32_grad = nullptr;
    platform::float16 *fused_fp16_param = nullptr, *fused_fp16_grad = nullptr;
    framework::Tensor *fp32_p_t = nullptr, *fp16_p_t = nullptr,
                      *fp32_g_t = nullptr, *fp16_g_t = nullptr;
    std::vector<framework::Tensor *> fp16_master_params;
    if (total_numel > 0) {
      fp32_p_t = ctx.Output<framework::Tensor>("FP32FusedParam");
      fused_fp32_param = TensorFillConstant<float>(
          dev_ctx, fp32_p_t, {static_cast<int64_t>(total_numel)}, 0.0f);
    }

    if (fp32_numel > 0) {
      fp32_g_t = ctx.Output<framework::Tensor>("FP32FusedGrad");
      fused_fp32_grad = TensorFillConstant<float>(
          dev_ctx, fp32_g_t, {static_cast<int64_t>(fp32_numel)}, 0.0f);
    }

    if (fp16_numel > 0) {
      fp16_p_t = ctx.Output<framework::Tensor>("FP16FusedParam");
      fused_fp16_param = TensorFillConstant<platform::float16>(
          dev_ctx, fp16_p_t, {static_cast<int64_t>(fp16_numel)},
          static_cast<platform::float16>(0));

      fp16_g_t = ctx.Output<framework::Tensor>("FP16FusedGrad");
      fused_fp16_grad = TensorFillConstant<platform::float16>(
          dev_ctx, fp16_g_t, {static_cast<int64_t>(fp16_numel)},
          static_cast<platform::float16>(0));
    }
    VLOG(10) << "Allocate FP32FusedParam/Grad, FP16FusedParam/Grad ends";

    // (1) For FP32FusedParam, memcpy for fp32 param and then share data, cast
    // for fp16 master weight
    // (2) For FP16FusedParam, memcpy and then share data
    // (3) For FP32FusedGrad/FP16FusedGrad, memcpy if gradient has been inited
    for (const auto &info : fp32_infos) {
      auto sliced_tensor = CopyAndShareBufferForInitedTensor(
          info.param_t, fp32_p_t, info.numel_offset, stream);
      master_params[info.idx]->Resize(info.param_t->dims());
      master_params[info.idx]->ShareBufferWith(sliced_tensor);
      PADDLE_ENFORCE_EQ(master_params[info.idx]->mutable_data<float>(place),
                        sliced_tensor.data<float>(),
                        platform::errors::InvalidArgument(
                            "Invalid master weight tensor pointer."));
      if (info.grad_t->IsInitialized()) {
        CopyAndShareBufferForInitedTensor(info.grad_t, fp32_g_t,
                                          info.numel_offset, stream);
      } else {
        ShareBufferForNonInitedTensor(info.grad_t, fp32_g_t, info.numel_offset,
                                      info.param_t->dims());
      }
    }

    size_t fp16_numel_offset = 0;
    if (fp32_numel > 0) {
      auto last_fp32_info = fp32_infos.back();
      fp16_numel_offset =
          last_fp32_info.numel_offset + last_fp32_info.numel_with_padding;
    }

    for (const auto &info : fp16_infos) {
      auto master_weight_offset = info.numel_offset + fp16_numel_offset;
      auto sliced_tensor = CastDataForInitedTensor(
          dev_ctx, info.param_t, fp32_p_t, master_weight_offset);
      master_params[info.idx]->Resize(info.param_t->dims());
      master_params[info.idx]->ShareBufferWith(sliced_tensor);

      CopyAndShareBufferForInitedTensor(info.param_t, fp16_p_t,
                                        info.numel_offset, stream);
      PADDLE_ENFORCE_EQ(master_params[info.idx]->mutable_data<float>(place),
                        sliced_tensor.data<float>(),
                        platform::errors::InvalidArgument(
                            "Invalid master weight tensor pointer."));

      if (info.grad_t->IsInitialized()) {
        CopyAndShareBufferForInitedTensor(info.grad_t, fp16_g_t,
                                          info.numel_offset, stream);
      } else {
        ShareBufferForNonInitedTensor(info.grad_t, fp16_g_t, info.numel_offset,
                                      info.param_t->dims());
      }
    }
    VLOG(10) << "Copy/share data for Param/Grad ends";

    // Step 4: For Moment1, Moment2, Beta1Pow, Beta2Pow, just fill constant
    TensorFillConstant<float>(dev_ctx, ctx.Output<framework::Tensor>("Moment1"),
                              {static_cast<int64_t>(numel_each_device)}, 0.0f);
    TensorFillConstant<float>(dev_ctx, ctx.Output<framework::Tensor>("Moment2"),
                              {static_cast<int64_t>(numel_each_device)}, 0.0f);
    TensorFillConstant<float>(dev_ctx,
                              ctx.Output<framework::Tensor>("Beta1Pow"), {1},
                              ctx.Attr<float>("beta1"));
    TensorFillConstant<float>(dev_ctx,
                              ctx.Output<framework::Tensor>("Beta2Pow"), {1},
                              ctx.Attr<float>("beta2"));
    VLOG(10) << "Init Moment and BetaPow ends";

    // Step 5: Do sharding
    size_t fp32_start_idx, fp32_end_idx, fp32_start_numel_offset,
        fp32_end_numel_offset;
    GetParamGradShardInfo(fp32_infos, rank * fp32_numel_each_device,
                          (rank + 1) * fp32_numel_each_device, &fp32_start_idx,
                          &fp32_end_idx, &fp32_start_numel_offset,
                          &fp32_end_numel_offset);
    size_t fp16_start_idx, fp16_end_idx, fp16_start_numel_offset,
        fp16_end_numel_offset;
    GetParamGradShardInfo(fp16_infos, rank * fp16_numel_each_device,
                          (rank + 1) * fp16_numel_each_device, &fp16_start_idx,
                          &fp16_end_idx, &fp16_start_numel_offset,
                          &fp16_end_numel_offset);
    size_t fp32_local_param_num =
        fp32_numel_each_device > 0 ? fp32_end_idx - fp32_start_idx + 1 : 0;
    size_t fp16_local_param_num =
        fp16_numel_each_device > 0 ? fp16_end_idx - fp16_start_idx + 1 : 0;
    size_t total_local_param_num = fp32_local_param_num + fp16_local_param_num;
    VLOG(10) << "Found the sharding arguments";

    auto *param_info_t = ctx.Output<framework::Tensor>("ParamInfo");
    param_info_t->Resize({8});
    auto *param_info = param_info_t->mutable_data<int>(platform::CPUPlace());
    param_info[0] = static_cast<int>(fp32_start_idx);
    param_info[1] = static_cast<int>(fp32_local_param_num);
    param_info[2] = static_cast<int>(fp32_infos.size());
    param_info[3] = ClipByBound<int>(fp32_wd_end_idx, fp32_start_idx,
                                     fp32_start_idx + fp32_local_param_num) -
                    static_cast<int>(fp32_start_idx);
    param_info[4] = static_cast<int>(fp16_start_idx + fp32_infos.size());
    param_info[5] = static_cast<int>(fp16_local_param_num);
    param_info[6] = static_cast<int>(fp16_infos.size());
    param_info[7] = ClipByBound<int>(fp16_wd_end_idx, fp16_start_idx,
                                     fp16_start_idx + fp16_local_param_num) -
                    static_cast<int>(fp16_start_idx);

    VLOG(10) << "Start FP32 idx: " << param_info[0];
    VLOG(10) << "Local FP32 param num: " << param_info[1];
    VLOG(10) << "Global FP32 param num: " << param_info[2];

    VLOG(10) << "Start FP16 idx: " << param_info[4];
    VLOG(10) << "Local FP16 param num: " << param_info[5];
    VLOG(10) << "Global FP16 param num: " << param_info[6];

    std::vector<int> numel_offsets;
    numel_offsets.reserve(params.size() + 1);
    for (const auto &info : fp32_infos) {
      numel_offsets.push_back(info.numel_offset);
    }
    for (const auto &info : fp16_infos) {
      numel_offsets.push_back(info.numel_offset + fp16_numel_offset);
    }
    numel_offsets.push_back(fp32_numel + fp16_numel);
    PADDLE_ENFORCE_EQ(numel_offsets.size(), params.size() + 1,
                      platform::errors::InvalidArgument(
                          "The numel_offsets number must be one larger than "
                          "the parameter number."));
    VLOG(10) << "Total numel offset: " << FlattenToString(numel_offsets);

    std::vector<int> fp32_partial_numel_offsets;
    fp32_partial_numel_offsets.reserve(fp32_local_param_num + 1);
    fp32_partial_numel_offsets.push_back(0);
    // Fill the partial_numel_offsets
    for (size_t i = fp32_start_idx; i < fp32_start_idx + fp32_local_param_num;
         ++i) {
      size_t valid_start_n = 0;
      if (i == fp32_start_idx) {
        valid_start_n = fp32_start_numel_offset;
      }

      size_t end_n = fp32_infos[i].numel_with_padding;
      if (i + 1 == fp32_start_idx + fp32_local_param_num) {
        end_n = std::min(end_n, fp32_end_numel_offset);
      }

      PADDLE_ENFORCE_NE(valid_start_n, end_n,
                        platform::errors::InvalidArgument(
                            "Indices sharding error. This may be a bug."));
      VLOG(10) << "FP32 Partial numel = ["
               << valid_start_n + fp32_infos[i].numel << ","
               << end_n + fp32_infos[i].numel;
      auto len = end_n - valid_start_n;
      fp32_partial_numel_offsets.push_back(fp32_partial_numel_offsets.back() +
                                           len);
    }

    std::vector<int> fp16_partial_numel_offsets;
    fp16_partial_numel_offsets.reserve(fp16_local_param_num + 1);
    fp16_partial_numel_offsets.push_back(0);
    for (size_t i = fp16_start_idx; i < fp16_start_idx + fp16_local_param_num;
         ++i) {
      size_t valid_start_n = 0;
      if (i == fp16_start_idx) {
        valid_start_n = fp16_start_numel_offset;
      }

      size_t end_n = fp16_infos[i].numel_with_padding;
      if (i + 1 == fp16_start_idx + fp16_local_param_num) {
        end_n = std::min(end_n, fp16_end_numel_offset);
      }

      PADDLE_ENFORCE_NE(valid_start_n, end_n,
                        platform::errors::InvalidArgument(
                            "Indices sharding error. This may be a bug."));
      auto len = end_n - valid_start_n;
      fp16_partial_numel_offsets.push_back(fp16_partial_numel_offsets.back() +
                                           len);
    }

    CopyVectorToCPUTensor(numel_offsets,
                          ctx.Output<framework::Tensor>("FusedParamOffsets"));
    CopyVectorToCPUTensor(
        fp32_partial_numel_offsets,
        ctx.Output<framework::Tensor>("FP32ShardFusedParamOffsets"));
    CopyVectorToCPUTensor(
        fp16_partial_numel_offsets,
        ctx.Output<framework::Tensor>("FP16ShardFusedParamOffsets"));

    auto *global_scale = ctx.Output<framework::Tensor>("GlobalScale");
    if (!global_scale->IsInitialized()) {
      TensorFillConstant<float>(dev_ctx, global_scale, {1}, 1.0f);
    }
    VLOG(10) << "Init global scale ends";
    dev_ctx.Wait();
    VLOG(10) << "Wait for H2D copy";
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(
    distributed_fused_lamb_init,
    ops::DistributedFusedLambInitOpKernel<plat::CUDADeviceContext, float>);
