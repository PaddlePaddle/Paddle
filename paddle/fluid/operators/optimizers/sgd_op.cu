#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/fluid/operators/optimizers/sgd_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

namespace {

template <typename T>
__global__ void SGDKernel(const T* g, const T* p, const T* learning_rate,
                          const int num, T* p_out) {
  T lr = learning_rate[0];
  int grid_size = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += grid_size) {
    T g_data = g[i];
    T p_data = p[i];
    p_out[i] = p_data - lr * g_data;
  }
}

template <typename T>
__global__ void SparseSGDFunctorKernel(const T* selected_rows,
                                       const int64_t* rows,
                                       const T* learning_rate, T* tensor_out,
                                       int64_t row_numel, int64_t limit) {
  for (int64_t i = blockIdx.x; i < limit; i += gridDim.x) {
    const T* selected_rows_ptr = selected_rows + i * row_numel;
    T* tensor_out_ptr = tensor_out + rows[i] * row_numel;
    for (int64_t index = threadIdx.x; index < row_numel; index += blockDim.x) {
      // Since index in rows of SelectedRows can be duplicate, we have to use
      // Atomic Operation to avoid concurrent write error.
      paddle::platform::CudaAtomicAdd(
          tensor_out_ptr + index,
          -static_cast<T>(1.0) * learning_rate[0] * selected_rows_ptr[index]);
    }
  }
}
}  // namespace

template <typename T>
class SGDOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const auto* param_var = ctx.InputVar("Param");
    PADDLE_ENFORCE(param_var->IsType<framework::LoDTensor>(),
                   "The Var(%s)'s type should be LoDTensor, "
                   "but the received is %s",
                   ctx.Inputs("Param").front(),
                   framework::ToTypeName(param_var->Type()));

    auto* param = ctx.Input<framework::Tensor>("Param");
    auto* param_out = ctx.Output<framework::Tensor>("ParamOut");
    auto* learning_rate = ctx.Input<framework::Tensor>("LearningRate");

    auto* grad_var = ctx.InputVar("Grad");
    // Actually, all tensors are LoDTensor except SelectedRows.
    if (grad_var->IsType<framework::LoDTensor>()) {
      param_out->mutable_data<T>(ctx.GetPlace());
      auto* grad = ctx.Input<framework::Tensor>("Grad");
      auto* grad_data = grad->data<T>();
      auto* param_data = param->data<T>();
      auto* param_out_data = param_out->data<T>();

      int block = 512;
      int grid = (param->numel() + block - 1) / block;

      SGDKernel<T><<<grid, block, 0, ctx.cuda_device_context().stream()>>>(
          grad_data, param_data, learning_rate->data<T>(), param->numel(),
          param_out_data);

    } else if (grad_var->IsType<framework::SelectedRows>()) {
      // TODO(qijun): In Sparse SGD operator, in-place update is enforced.
      // This manual optimization brings difficulty to track data dependency.
      // It's better to find a more elegant solution.
      PADDLE_ENFORCE_EQ(param, param_out);
      auto* grad = ctx.Input<framework::SelectedRows>("Grad");

      auto in_height = grad->height();
      auto out_dims = param_out->dims();
      PADDLE_ENFORCE_EQ(in_height, out_dims[0]);

      auto& in_value = grad->value();
      auto& in_rows = grad->rows();

      int64_t in_row_numel = in_value.numel() / in_rows.size();
      PADDLE_ENFORCE_EQ(in_row_numel, param_out->numel() / in_height);

      auto* in_data = in_value.data<T>();
      auto* out_data = param_out->data<T>();

      const int kThreadsPerBlock = 256;
      int thread_x = kThreadsPerBlock;
      int max_threads = ctx.cuda_device_context().GetMaxPhysicalThreadCount();
      int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

      SparseSGDFunctorKernel<<<max_blocks, thread_x, 0,
                               ctx.cuda_device_context().stream()>>>(
          in_data, in_rows.CUDAData(ctx.GetPlace()), learning_rate->data<T>(),
          out_data, in_row_numel, in_rows.size());

    } else {
      PADDLE_THROW("Unsupported Variable Type of Grad");
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
REGISTER_OP_CUDA_KERNEL(sgd, ops::SGDOpCUDAKernel<float>,
                        ops::SGDOpCUDAKernel<double>,
                        ops::SGDOpCUDAKernel<plat::float16>);
