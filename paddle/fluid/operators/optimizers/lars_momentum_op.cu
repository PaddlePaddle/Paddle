#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/operators/optimizers/lars_momentum_op.h"
#include "paddle/fluid/platform/fast_divmod.h"

#if defined(__NVCC__) && CUDA_VERSION >= 11000
#include <hip/hip_cooperative_groups.h>
#define LARS_FUNCTION_FLAG __device__
#else
#define LARS_FUNCTION_FLAG __global__
#endif

#ifdef __HIPCC__
#define LARS_BLOCK_SIZE 256
#else
#define LARS_BLOCK_SIZE 512
#endif

namespace paddle {
namespace operators {

template <typename T>
using MultiPrecisionType = typename details::MPTypeTrait<T>::Type;

template <typename MT, int VecSize>
__device__ inline void VectorizeLarsUpdate(
    const MT* __restrict__ g, const MT* __restrict__ v, MT* __restrict__ p_out,
    MT* __restrict__ v_out, const MT* __restrict__ p, const MT mu, MT local_lr,
    const MT lars_weight_decay, const MT rescale_grad, const int tid,
    const int grid_stride, const int numel) {
  using VecMType = paddle::platform::AlignedVector<MT, VecSize>;
  int main = numel >> (VecSize >> 1);
  int tail_offset = main * VecSize;

  const VecMType* __restrict__ g_vec = reinterpret_cast<const VecMType*>(g);
  const VecMType* __restrict__ v_vec = reinterpret_cast<const VecMType*>(v);
  const VecMType* __restrict__ p_vec = reinterpret_cast<const VecMType*>(p);
  VecMType* p_out_vec = reinterpret_cast<VecMType*>(p_out);
  VecMType* v_out_vec = reinterpret_cast<VecMType*>(v_out);

  for (int i = tid; i < main; i += grid_stride) {
    VecMType v_new, p_new;
    VecMType g_data = g_vec[i];
    VecMType v_data = v_vec[i];
    VecMType p_data = p_vec[i];

#pragma unroll
    for (int j = 0; j < VecSize; ++j) {
      MT grad = g_data.val[j] * rescale_grad;
      v_new.val[j] =
          fma(v_data.val[j], mu,
              local_lr * fma(lars_weight_decay, p_data.val[j], grad));
      p_new.val[j] = p_data.val[j] - v_new.val[j];
    }
    v_out_vec[i] = v_new;
    p_out_vec[i] = p_new;
  }

  for (int i = tid + tail_offset; i < numel; i += grid_stride) {
    MT grad = g[i] * rescale_grad;
    MT param = p[i];
    MT v_new = fma(v[i], mu, local_lr * fma(lars_weight_decay, param, grad));
    v_out[i] = v_new;
    p_out[i] = param - v_new;
  }
}

template <typename T, typename MT>
__device__ inline void VectorizeLarsUpdateMP(
    const T* __restrict__ g, const MT* __restrict__ v, T* __restrict__ p_out,
    MT* __restrict__ v_out, const MT* __restrict__ master_p,
    MT* __restrict__ master_p_out, const MT mu, MT local_lr,
    const MT lars_weight_decay, const MT rescale_grad, const int tid,
    const int grid_stride, const int numel) {
  // As for multiple-precision, type T and MT cannot be more than fp16 or fp32,
  // Then, the maximum data IO size could be set to 4.
  using VecType = paddle::platform::AlignedVector<T, 4>;
  using VecMType = paddle::platform::AlignedVector<MT, 4>;
  int main = numel >> 2;
  int tail_offset = main << 2;

  const VecType* __restrict__ g_vec = reinterpret_cast<const VecType*>(g);
  const VecMType* __restrict__ v_vec = reinterpret_cast<const VecMType*>(v);
  const VecMType* __restrict__ master_p_vec =
      reinterpret_cast<const VecMType*>(master_p);
  VecType* p_out_vec = reinterpret_cast<VecType*>(p_out);
  VecMType* v_out_vec = reinterpret_cast<VecMType*>(v_out);
  VecMType* master_p_out_vec = reinterpret_cast<VecMType*>(master_p_out);

  for (int i = tid; i < main; i += grid_stride) {
    VecType p_out;
    VecMType v_new, p_new;
    VecType g_data = g_vec[i];
    VecMType v_data = v_vec[i];
    VecMType p_data = master_p_vec[i];

#pragma unroll
    for (int j = 0; j < 4; ++j) {
      MT grad = static_cast<MT>(g_data.val[j]) * rescale_grad;
      v_new.val[j] =
          fma(v_data.val[j], mu,
              local_lr * fma(lars_weight_decay, p_data.val[j], grad));
      p_new.val[j] = p_data.val[j] - v_new.val[j];
      p_out.val[j] = static_cast<T>(p_new.val[j]);
    }
    v_out_vec[i] = v_new;
    p_out_vec[i] = p_out;
    master_p_out_vec[i] = p_new;
  }

  for (int i = tid + tail_offset; i < numel; i += grid_stride) {
    MT grad = static_cast<MT>(g[i]) * rescale_grad;
    MT param = master_p[i];
    MT v_new = fma(v[i], mu, local_lr * fma(lars_weight_decay, param, grad));
    MT p_new = param - v_new;
    v_out[i] = v_new;
    p_out[i] = static_cast<T>(p_new);
    master_p_out[i] = p_new;
  }
}

template <typename T, typename MT>
LARS_FUNCTION_FLAG void L2NormKernel(
    const T* __restrict__ p_data, const T* __restrict__ g_data,
    MT* __restrict__ p_buffer, MT* __restrict__ g_buffer,
    const int repeat_times, const int64_t numel, const MT rescale_grad,
    MT* __restrict__ p_n = nullptr, MT* __restrict__ g_n = nullptr) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int grid_stride = LARS_BLOCK_SIZE * gridDim.x;
  const MT rescale_grad_pow = rescale_grad * rescale_grad;
  __shared__ MT s_buffer[2];
  s_buffer[0] = static_cast<MT>(0);
  s_buffer[1] = static_cast<MT>(0);
  MT p_tmp_val = static_cast<MT>(0);
  MT g_tmp_val = static_cast<MT>(0);

  if (repeat_times == 0) {
    if (tid < numel) {
      p_tmp_val = static_cast<MT>(p_data[tid]);
      g_tmp_val = static_cast<MT>(g_data[tid]);
    }
    s_buffer[0] += math::blockReduceSum<MT>(p_tmp_val * p_tmp_val, FINAL_MASK);
    s_buffer[1] += math::blockReduceSum<MT>(g_tmp_val * g_tmp_val, FINAL_MASK);
  } else {
    /* To avoid occupy too much temp buffer. Hence, slice the whole data into 2
    parts, the front of them whose quantity is excatly multiple of grid-thread
    number, and this part of data is delt in for loop, the rest of data is delt
    with another step to avoid visiting data address beyond bound. */
    for (int i = 0; i < repeat_times; ++i) {
      p_tmp_val = static_cast<MT>(p_data[tid]);
      g_tmp_val = static_cast<MT>(g_data[tid]);
      tid += grid_stride;
      s_buffer[0] +=
          math::blockReduceSum<MT>(p_tmp_val * p_tmp_val, FINAL_MASK);
      s_buffer[1] +=
          math::blockReduceSum<MT>(g_tmp_val * g_tmp_val, FINAL_MASK);
      __syncthreads();
    }
    MT p_val = 0;
    MT g_val = 0;
    if (tid < numel) {
      p_val = static_cast<MT>(p_data[tid]);
      g_val = static_cast<MT>(g_data[tid]);
    }
    s_buffer[0] += math::blockReduceSum<MT>(p_val * p_val, FINAL_MASK);
    s_buffer[1] += math::blockReduceSum<MT>(g_val * g_val, FINAL_MASK);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    p_buffer[blockIdx.x] = s_buffer[0];
    g_buffer[blockIdx.x] = rescale_grad_pow * s_buffer[1];
  }

#if CUDA_VERSION >= 11000
  // Grid sync for completely writring partial result back to gloabl memory
  const cooperative_groups::grid_group cg = cooperative_groups::this_grid();
  cg.sync();
  MT p_partial_sum = threadIdx.x < gridDim.x ? p_buffer[threadIdx.x] : 0;
  MT g_partial_sum = threadIdx.x < gridDim.x ? g_buffer[threadIdx.x] : 0;
  *p_n = sqrt(math::blockReduceSum<MT>(p_partial_sum, FINAL_MASK));
  *g_n = sqrt(math::blockReduceSum<MT>(g_partial_sum, FINAL_MASK));
#endif
}

template <typename T, typename MT>
__global__ void MomentumLarsKernel(
    const T* __restrict__ p, const T* __restrict__ g, const MT* __restrict__ v,
    T* p_out, MT* v_out, const MT* __restrict__ master_p,
    MT* __restrict__ master_p_out, const MT* __restrict__ learning_rate,
    MT* __restrict__ p_buffer, MT* __restrict__ g_buffer, const MT mu,
    const MT lars_coeff, const MT lars_weight_decay, const MT epsilon,
    const MT rescale_grad, const int repeat_times, const int thresh,
    const int64_t numel) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int grid_stride = gridDim.x * LARS_BLOCK_SIZE;
#if CUDA_VERSION >= 11000
  MT p_n = static_cast<MT>(0);
  MT g_n = static_cast<MT>(0);
  L2NormKernel<T, MT>(p, g, p_buffer, g_buffer, repeat_times, numel,
                      rescale_grad, &p_n, &g_n);
#else
  MT p_val = threadIdx.x < thresh ? p_buffer[threadIdx.x] : 0;
  MT g_val = threadIdx.x < thresh ? g_buffer[threadIdx.x] : 0;
  __syncthreads();
  MT p_n = sqrt(math::blockReduceSum<MT>(p_val, FINAL_MASK));
  MT g_n = sqrt(math::blockReduceSum<MT>(g_val, FINAL_MASK));
#endif

  const MT lr = learning_rate[0];
  MT local_lr = lr;
  if (lars_weight_decay > static_cast<MT>(0)) {
    local_lr =
        lr * lars_coeff * p_n / (fma(lars_weight_decay, p_n, g_n) + epsilon);
  }

  if (master_p) {
    VectorizeLarsUpdateMP<T, MT>(g, v, p_out, v_out, master_p, master_p_out, mu,
                                 local_lr, lars_weight_decay, rescale_grad, tid,
                                 grid_stride, numel);
  } else {
    if (std::is_same<T, float>::value ||
        std::is_same<T, paddle::platform::float16>::value) {
      VectorizeLarsUpdate<MT, 4>(
          reinterpret_cast<const MT*>(g), v, reinterpret_cast<MT*>(p_out),
          v_out, reinterpret_cast<const MT*>(p), mu, local_lr,
          lars_weight_decay, rescale_grad, tid, grid_stride, numel);
    } else {
      VectorizeLarsUpdate<MT, 2>(
          reinterpret_cast<const MT*>(g), v, reinterpret_cast<MT*>(p_out),
          v_out, reinterpret_cast<const MT*>(p), mu, local_lr,
          lars_weight_decay, rescale_grad, tid, grid_stride, numel);
    }
  }
}

template <typename DeviceContext, typename T>
class LarsMomentumOpCUDAKernel : public framework::OpKernel<T> {
  using MT = MultiPrecisionType<T>;

 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const bool multi_precision = ctx.Attr<bool>("multi_precision");
    auto param_out = ctx.Output<framework::LoDTensor>("ParamOut");
    auto velocity_out = ctx.Output<framework::LoDTensor>("VelocityOut");
    auto param = ctx.Input<framework::LoDTensor>("Param");
    auto velocity = ctx.Input<framework::LoDTensor>("Velocity");
    auto grad = ctx.Input<framework::LoDTensor>("Grad");
    auto learning_rate = ctx.Input<framework::LoDTensor>("LearningRate");

    int64_t numel = param->numel();
    int grid = (numel + LARS_BLOCK_SIZE - 1) / LARS_BLOCK_SIZE;
    const framework::Tensor* master_param = nullptr;
    framework::Tensor* master_param_out = nullptr;
    const MT* master_p = nullptr;
    MT* master_p_out = nullptr;

    if (multi_precision) {
      bool has_master =
          ctx.HasInput("MasterParam") && ctx.HasOutput("MasterParamOut");
      PADDLE_ENFORCE_EQ(has_master, true,
                        platform::errors::InvalidArgument(
                            "The Input(MasterParam) and Output(MasterParamOut) "
                            "should not be null when "
                            "the attr `multi_precision` is true"));
      master_param = ctx.Input<framework::Tensor>("MasterParam");
      master_param_out = ctx.Output<framework::Tensor>("MasterParamOut");
      master_p = master_param->data<MT>();
      master_p_out = master_param_out->mutable_data<MT>(ctx.GetPlace());
    }
    T* p_out = param_out->mutable_data<T>(ctx.GetPlace());
    MT* v_out = velocity_out->mutable_data<MT>(ctx.GetPlace());

    MT mu = static_cast<MT>(ctx.Attr<float>("mu"));
    MT lars_coeff = static_cast<MT>(ctx.Attr<float>("lars_coeff"));
    MT lars_weight_decay =
        static_cast<MT>(ctx.Attr<float>("lars_weight_decay"));
    MT epsilon = static_cast<MT>(ctx.Attr<float>("epsilon"));
    MT rescale_grad = static_cast<MT>(ctx.Attr<float>("rescale_grad"));

    auto* p = param->data<T>();
    auto* g = grad->data<T>();
    auto* v = velocity->data<MT>();
    auto* lr = learning_rate->data<MT>();
    auto& cuda_ctx = ctx.template device_context<platform::CUDADeviceContext>();

#if CUDA_VERSION >= 11000
    /*
    Once model trainning with lars optimizer, whose principal implementation
    is achieved by following two steps:
      1. Figure out the L2 norm statistic result of grad data and param data.
      2. Update param and velocity data with usage of L2 norm statistic result.

    Orignally, these two steps were fulfilled by respective eigen function and
    cuda kernel, however the overhead of eigen function occupied much ratio in
    total, consequently affect the performance of lars op, make it necessary
    to combine 2 steps into one cuda kernel.
    Since the step1 is l2 norm statistic, grid level reduce is needed. To
    achieve this and continuous calculation of step 2 in only one global
    lanuch, essential basis is to control all grid-threads while running. Apart
    from normal lanuch form, cuda9.0 provides `hipLaunchCooperativeKernel`
    api :
      - The thread quantity shall less than pyhsical SM limited threads
      - Launches a device function where thread blocks can cooperate and
        synchronize as they execute.
    */
    // Figure out how many blocks can be active in each sm.
    int num_blocks_per_sm = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm,
                                                  MomentumLarsKernel<T, MT>,
                                                  LARS_BLOCK_SIZE, sizeof(MT));
    int sm_num = cuda_ctx.GetSMCount();
    int grid_real =
        std::min(std::min(sm_num * num_blocks_per_sm, grid), LARS_BLOCK_SIZE);
    framework::Tensor tmp_buffer_t =
        ctx.AllocateTmpTensor<MT, platform::CUDADeviceContext>(
            {LARS_BLOCK_SIZE << 1}, cuda_ctx);
    auto* p_buffer = tmp_buffer_t.mutable_data<MT>(ctx.GetPlace());
    auto* g_buffer = p_buffer + LARS_BLOCK_SIZE;
    int grid_stride = LARS_BLOCK_SIZE * grid;
    int repeat_times = (numel + grid_stride - 1) / grid_stride - 1;
    int thresh = 0;

    // Uniform kernel parameter for hipLaunchCooperativeKernel
    void* cuda_param[] = {
        reinterpret_cast<void*>(&p),
        reinterpret_cast<void*>(&g),
        reinterpret_cast<void*>(&v),
        reinterpret_cast<void*>(&p_out),
        reinterpret_cast<void*>(&v_out),
        reinterpret_cast<void*>(&master_p),
        reinterpret_cast<void*>(&master_p_out),
        reinterpret_cast<void*>(&lr),
        reinterpret_cast<void*>(&p_buffer),
        reinterpret_cast<void*>(&g_buffer),
        reinterpret_cast<void*>(&mu),
        reinterpret_cast<void*>(&lars_coeff),
        reinterpret_cast<void*>(&lars_weight_decay),
        reinterpret_cast<void*>(&epsilon),
        reinterpret_cast<void*>(&rescale_grad),
        reinterpret_cast<void*>(&repeat_times),
        reinterpret_cast<void*>(&thresh),  // Just a placeholder
        reinterpret_cast<void*>(&numel)};
    // Lanuch all sm theads.
    hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
        reinterpret_cast<void*>(MomentumLarsKernel<T), MT>), grid_real,
        LARS_BLOCK_SIZE, cuda_param, 0, cuda_ctx.stream());
#else
    // Determine to read 4 fp16 or float data once, but 2 double data once.
    int grid_lars =
        sizeof(T) < 64
            ? (numel + (LARS_BLOCK_SIZE << 2) - 1) / (LARS_BLOCK_SIZE << 2)
            : (numel + (LARS_BLOCK_SIZE << 1) - 1) / (LARS_BLOCK_SIZE << 1);

    int grid_norm = std::min(grid, LARS_BLOCK_SIZE);
    framework::Tensor p_buffer_t =
        ctx.AllocateTmpTensor<MT, platform::CUDADeviceContext>(
            {LARS_BLOCK_SIZE << 1}, cuda_ctx);
    auto* p_buffer = p_buffer_t.mutable_data<MT>(ctx.GetPlace());
    auto* g_buffer = p_buffer + LARS_BLOCK_SIZE;

    const int grid_stride = LARS_BLOCK_SIZE * grid_norm;
    const int repeat_times = (numel + grid_stride - 1) / grid_stride - 1;

    L2NormKernel<T, MT><<<grid_norm, LARS_BLOCK_SIZE, 0, cuda_ctx.stream()>>>(
        p, g, p_buffer, g_buffer, repeat_times, numel, rescale_grad);

    MomentumLarsKernel<
        T, MT><<<grid_lars, LARS_BLOCK_SIZE, 0, cuda_ctx.stream()>>>(
        p, g, v, p_out, v_out, master_p, master_p_out, lr, p_buffer, g_buffer,
        mu, lars_coeff, lars_weight_decay, epsilon, rescale_grad, 0, grid_norm,
        numel);  // 0 is just a placeholder.
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    lars_momentum,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>);
