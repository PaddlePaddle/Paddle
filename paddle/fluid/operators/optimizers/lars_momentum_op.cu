#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/operators/optimizers/lars_momentum_op.h"
#include "paddle/fluid/platform/fast_divmod.h"

#if defined(__NVCC__) && CUDA_VERSION >= 11000
/* Once CUDA_VERSION is beyond 11.0, cooperative_groups can be involved in
   without adding --rdc=true compile flag, then L2_norm cuda kernel can be
   set as a __device__ kernel rather than global kernel. On the contrary,
   the compile flag shall be set in old version, which may affect the cuda
   kernel performance in paddle, consequently, L2_norm kernel shall be set
   as a __global__ kernel.
*/
#include <hip/hip_cooperative_groups.h>
#define LARS_FUNCTION_FLAG __device__
#else
#define LARS_FUNCTION_FLAG __global__
#endif

#ifdef __HIPCC__
#define LARS_BLOCK_SIZE 256
#else
#define LARS_BLOCK_SIZE 512
#endif

namespace paddle {
namespace operators {

template <typename T>
using MultiPrecisionType = typename details::MPTypeTrait<T>::Type;

__device__ __forceinline__ float Sqrt(float x) { return sqrtf(x); }
__device__ __forceinline__ double Sqrt(double x) { return sqrt(x); }
__device__ __forceinline__ float Fma(float x, float y, float z) {
  return fmaf(x, y, z);
}
__device__ __forceinline__ double Fma(double x, double y, double z) {
  return fma(x, y, z);
}

template <typename T, typename MT, int VecSize, bool IsAmp = false>
__device__ inline void VectorizeLarsUpdate(
    const T* __restrict__ grad, const MT* __restrict__ param,
    const MT* __restrict__ velocity, T* __restrict__ param_out,
    MT* __restrict__ velocity_out, const MT mu, MT local_lr,
    const MT lars_weight_decay, const MT rescale_grad, const int tid,
    const int grid_stride, const int numel,
    MT* __restrict__ master_param_out = nullptr) {
  using VecType = paddle::platform::AlignedVector<T, VecSize>;
  using VecMType = paddle::platform::AlignedVector<MT, VecSize>;
  int main = numel >> (VecSize >> 1);
  int tail_offset = main * VecSize;

  const VecType* __restrict__ grad_vec = reinterpret_cast<const VecType*>(grad);
  const VecMType* __restrict__ param_vec =
      reinterpret_cast<const VecMType*>(param);
  const VecMType* __restrict__ velocity_vec =
      reinterpret_cast<const VecMType*>(velocity);
  VecType* param_out_vec = reinterpret_cast<VecType*>(param_out);
  VecMType* velocity_out_vec = reinterpret_cast<VecMType*>(velocity_out);

  VecMType* master_param_out_vec;
  if (IsAmp) {
    master_param_out_vec = reinterpret_cast<VecMType*>(master_param_out);
  }

  for (int i = tid; i < main; i += grid_stride) {
    VecType param_out_tmp;
    VecMType velocity_tmp, param_tmp;
    VecType grad_data = grad_vec[i];
    VecMType param_data = param_vec[i];
    VecMType velocity_data = velocity_vec[i];

#pragma unroll
    for (int j = 0; j < VecSize; ++j) {
      MT grad_val = static_cast<MT>(grad_data[j]) * rescale_grad;
      velocity_tmp[j] =
          Fma(velocity_data[j], mu,
              local_lr * Fma(lars_weight_decay, param_data[j], grad_val));
      param_tmp[j] = param_data[j] - velocity_tmp[j];
      param_out_tmp[j] = static_cast<T>(param_tmp[j]);
    }
    param_out_vec[i] = param_out_tmp;
    velocity_out_vec[i] = velocity_tmp;
    if (IsAmp) {
      master_param_out_vec[i] = param_tmp;
    }
  }

  for (int i = tid + tail_offset; i < numel; i += grid_stride) {
    MT grad_val = static_cast<MT>(grad[i]) * rescale_grad;
    MT param_val = param[i];
    MT velocity_tmp = Fma(velocity[i], mu, local_lr * Fma(lars_weight_decay,
                                                          param_val, grad_val));
    MT param_tmp = param_val - velocity_tmp;
    param_out[i] = static_cast<T>(param_tmp);
    velocity_out[i] = velocity_tmp;
    if (IsAmp) {
      master_param_out[i] = param_tmp;
    }
  }
}

template <typename T, typename MT>
LARS_FUNCTION_FLAG void L2NormKernel(
    const T* __restrict__ p_data, const T* __restrict__ g_data,
    MT* __restrict__ p_buffer, MT* __restrict__ g_buffer,
    const int repeat_times, const int64_t numel, const MT rescale_grad,
    MT* __restrict__ p_n = nullptr, MT* __restrict__ g_n = nullptr) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int grid_stride = LARS_BLOCK_SIZE * gridDim.x;
  const MT rescale_grad_pow = rescale_grad * rescale_grad;
  __shared__ MT s_buffer[2];
  s_buffer[0] = static_cast<MT>(0);
  s_buffer[1] = static_cast<MT>(0);
  MT p_tmp_val = static_cast<MT>(0);
  MT g_tmp_val = static_cast<MT>(0);

  if (repeat_times == 0) {
    if (tid < numel) {
      p_tmp_val = static_cast<MT>(p_data[tid]);
      g_tmp_val = static_cast<MT>(g_data[tid]);
    }
    s_buffer[0] += math::blockReduceSum<MT>(p_tmp_val * p_tmp_val, FINAL_MASK);
    s_buffer[1] += math::blockReduceSum<MT>(g_tmp_val * g_tmp_val, FINAL_MASK);
  } else {
    /* To avoid occupy too much temp buffer. Hence, slice the whole data into 2
    parts, the front of them whose quantity is excatly multiple of grid-thread
    number, and this part of data is delt in for loop, the rest of data is delt
    with another step to avoid visiting data address beyond bound. */
    for (int i = 0; i < repeat_times; ++i) {
      p_tmp_val = static_cast<MT>(p_data[tid]);
      g_tmp_val = static_cast<MT>(g_data[tid]);
      tid += grid_stride;
      s_buffer[0] +=
          math::blockReduceSum<MT>(p_tmp_val * p_tmp_val, FINAL_MASK);
      s_buffer[1] +=
          math::blockReduceSum<MT>(g_tmp_val * g_tmp_val, FINAL_MASK);
      __syncthreads();
    }
    MT p_val = 0;
    MT g_val = 0;
    if (tid < numel) {
      p_val = static_cast<MT>(p_data[tid]);
      g_val = static_cast<MT>(g_data[tid]);
    }
    s_buffer[0] += math::blockReduceSum<MT>(p_val * p_val, FINAL_MASK);
    s_buffer[1] += math::blockReduceSum<MT>(g_val * g_val, FINAL_MASK);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    p_buffer[blockIdx.x] = s_buffer[0];
    g_buffer[blockIdx.x] = rescale_grad_pow * s_buffer[1];
  }

#if CUDA_VERSION >= 11000
  // Grid sync for completely writring partial result back to gloabl memory
  const cooperative_groups::grid_group cg = cooperative_groups::this_grid();
  cg.sync();
  MT p_partial_sum = threadIdx.x < gridDim.x ? p_buffer[threadIdx.x] : 0;
  MT g_partial_sum = threadIdx.x < gridDim.x ? g_buffer[threadIdx.x] : 0;
  *p_n = Sqrt(math::blockReduceSum<MT>(p_partial_sum, FINAL_MASK));
  *g_n = Sqrt(math::blockReduceSum<MT>(g_partial_sum, FINAL_MASK));
#endif
}

template <typename T, typename MT>
__global__ void MomentumLarsKernel(
    const T* __restrict__ param, const T* __restrict__ grad,
    const MT* __restrict__ velocity, T* param_out, MT* velocity_out,
    const MT* __restrict__ master_param, MT* __restrict__ master_param_out,
    const MT* __restrict__ learning_rate, MT* __restrict__ p_buffer,
    MT* __restrict__ g_buffer, const MT mu, const MT lars_coeff,
    const MT lars_weight_decay, const MT epsilon, const MT rescale_grad,
    const int repeat_times, const int thresh, const int64_t numel) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int grid_stride = gridDim.x * LARS_BLOCK_SIZE;
#if CUDA_VERSION >= 11000
  MT param_norm = static_cast<MT>(0);
  MT grad_norm = static_cast<MT>(0);
  L2NormKernel<T, MT>(param, grad, p_buffer, g_buffer, repeat_times, numel,
                      rescale_grad, &param_norm, &grad_norm);
#else
  MT param_parital_norm = threadIdx.x < thresh ? p_buffer[threadIdx.x] : 0;
  MT grad_parital_norm = threadIdx.x < thresh ? g_buffer[threadIdx.x] : 0;
  __syncthreads();
  MT param_norm =
      Sqrt(math::blockReduceSum<MT>(param_parital_norm, FINAL_MASK));
  MT grad_norm = Sqrt(math::blockReduceSum<MT>(grad_parital_norm, FINAL_MASK));
#endif

  const MT lr = learning_rate[0];
  MT local_lr = lr;
  if (lars_weight_decay > static_cast<MT>(0)) {
    local_lr = lr * lars_coeff * param_norm /
               (Fma(lars_weight_decay, param_norm, grad_norm) + epsilon);
  }

  if (master_param_out) {
    VectorizeLarsUpdate<T, MT, 4, true>(grad, master_param, velocity, param_out,
                                        velocity_out, mu, local_lr,
                                        lars_weight_decay, rescale_grad, tid,
                                        grid_stride, numel, master_param_out);
  } else {
    if (std::is_same<T, float>::value ||
        std::is_same<T, paddle::platform::float16>::value) {
      // As for multiple-precision, type T and MT cannot be more than fp16 or
      // fp32, Then, the maximum data IO size could be set to 4.
      VectorizeLarsUpdate<T, MT, 4, false>(
          grad, reinterpret_cast<const MT*>(param), velocity, param_out,
          velocity_out, mu, local_lr, lars_weight_decay, rescale_grad, tid,
          grid_stride, numel);
    } else {
      VectorizeLarsUpdate<T, MT, 2, false>(
          grad, reinterpret_cast<const MT*>(param), velocity, param_out,
          velocity_out, mu, local_lr, lars_weight_decay, rescale_grad, tid,
          grid_stride, numel);
    }
  }
}

template <typename DeviceContext, typename T>
class LarsMomentumOpCUDAKernel : public framework::OpKernel<T> {
  using MT = MultiPrecisionType<T>;

 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const bool multi_precision = ctx.Attr<bool>("multi_precision");
    auto param_out = ctx.Output<framework::LoDTensor>("ParamOut");
    auto velocity_out = ctx.Output<framework::LoDTensor>("VelocityOut");
    auto param = ctx.Input<framework::LoDTensor>("Param");
    auto velocity = ctx.Input<framework::LoDTensor>("Velocity");
    auto grad = ctx.Input<framework::LoDTensor>("Grad");
    auto learning_rate = ctx.Input<framework::LoDTensor>("LearningRate");

    int64_t numel = param->numel();
    int grid = (numel + LARS_BLOCK_SIZE - 1) / LARS_BLOCK_SIZE;
    const framework::Tensor* master_param = nullptr;
    framework::Tensor* master_param_out = nullptr;
    const MT* master_param_data = nullptr;
    MT* master_param_out_data = nullptr;

    if (multi_precision) {
      bool has_master =
          ctx.HasInput("MasterParam") && ctx.HasOutput("MasterParamOut");
      PADDLE_ENFORCE_EQ(has_master, true,
                        platform::errors::InvalidArgument(
                            "The Input(MasterParam) and Output(MasterParamOut) "
                            "should not be null when "
                            "the attr `multi_precision` is true"));
      master_param = ctx.Input<framework::Tensor>("MasterParam");
      master_param_out = ctx.Output<framework::Tensor>("MasterParamOut");
      master_param_data = master_param->data<MT>();
      master_param_out_data =
          master_param_out->mutable_data<MT>(ctx.GetPlace());
    }
    MT mu = static_cast<MT>(ctx.Attr<float>("mu"));
    MT lars_coeff = static_cast<MT>(ctx.Attr<float>("lars_coeff"));
    MT lars_weight_decay =
        static_cast<MT>(ctx.Attr<float>("lars_weight_decay"));
    MT epsilon = static_cast<MT>(ctx.Attr<float>("epsilon"));
    MT rescale_grad = static_cast<MT>(ctx.Attr<float>("rescale_grad"));

    auto* param_data = param->data<T>();
    auto* grad_data = grad->data<T>();
    auto* velocity_data = velocity->data<MT>();
    auto* lr = learning_rate->data<MT>();
    auto& cuda_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    T* param_out_data = param_out->mutable_data<T>(ctx.GetPlace());
    MT* velocity_out_data = velocity_out->mutable_data<MT>(ctx.GetPlace());

#if CUDA_VERSION >= 11000
    /*
    Once model trainning with lars optimizer, whose principal implementation
    is achieved by following two steps:
      1. Figure out the L2 norm statistic result of grad data and param data.
      2. Update param and velocity data with usage of L2 norm statistic result.

    Orignally, these two steps were fulfilled by respective eigen function and
    cuda kernel, however the overhead of eigen function occupied much ratio in
    total, consequently affect the performance of lars op, make it necessary
    to combine 2 steps into one cuda kernel.
    Since the step1 is l2 norm statistic, grid level reduce is needed. To
    achieve this and continuous calculation of step 2 in only one global
    lanuch, essential basis is to control all grid-threads while running. Apart
    from normal lanuch form, cuda9.0 provides `hipLaunchCooperativeKernel`
    api :
      - The thread quantity shall less than pyhsical SM limited threads
      - Launches a device function where thread blocks can cooperate and
        synchronize as they execute.
    */
    // Figure out how many blocks can be active in each sm.
    int num_blocks_per_sm = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm,
                                                  MomentumLarsKernel<T, MT>,
                                                  LARS_BLOCK_SIZE, sizeof(MT));
    int sm_num = cuda_ctx.GetSMCount();
    int grid_real =
        std::min(std::min(sm_num * num_blocks_per_sm, grid), LARS_BLOCK_SIZE);
    framework::Tensor tmp_buffer_t =
        ctx.AllocateTmpTensor<MT, platform::CUDADeviceContext>(
            {LARS_BLOCK_SIZE << 1}, cuda_ctx);
    auto* p_buffer = tmp_buffer_t.mutable_data<MT>(ctx.GetPlace());
    auto* g_buffer = p_buffer + LARS_BLOCK_SIZE;
    int grid_stride = LARS_BLOCK_SIZE * grid;
    int repeat_times = (numel + grid_stride - 1) / grid_stride - 1;
    int thresh = 0;

    // Uniform kernel parameter for hipLaunchCooperativeKernel
    void* cuda_param[] = {
        reinterpret_cast<void*>(&param_data),
        reinterpret_cast<void*>(&grad_data),
        reinterpret_cast<void*>(&velocity_data),
        reinterpret_cast<void*>(&param_out_data),
        reinterpret_cast<void*>(&velocity_out_data),
        reinterpret_cast<void*>(&master_param_data),
        reinterpret_cast<void*>(&master_param_out_data),
        reinterpret_cast<void*>(&lr),
        reinterpret_cast<void*>(&p_buffer),
        reinterpret_cast<void*>(&g_buffer),
        reinterpret_cast<void*>(&mu),
        reinterpret_cast<void*>(&lars_coeff),
        reinterpret_cast<void*>(&lars_weight_decay),
        reinterpret_cast<void*>(&epsilon),
        reinterpret_cast<void*>(&rescale_grad),
        reinterpret_cast<void*>(&repeat_times),
        reinterpret_cast<void*>(&thresh),  // Just a placeholder
        reinterpret_cast<void*>(&numel)};
    // Lanuch all sm theads.
    hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
        reinterpret_cast<void*>(MomentumLarsKernel<T), MT>), grid_real,
        LARS_BLOCK_SIZE, cuda_param, 0, cuda_ctx.stream());
#else
    // Determine to read 4 fp16 or float data once, but 2 double data once.
    int grid_lars =
        sizeof(T) < sizeof(double)
            ? (numel + (LARS_BLOCK_SIZE << 2) - 1) / (LARS_BLOCK_SIZE << 2)
            : (numel + (LARS_BLOCK_SIZE << 1) - 1) / (LARS_BLOCK_SIZE << 1);

    int grid_norm = std::min(grid, LARS_BLOCK_SIZE);
    framework::Tensor p_buffer_t =
        ctx.AllocateTmpTensor<MT, platform::CUDADeviceContext>(
            {LARS_BLOCK_SIZE << 1}, cuda_ctx);
    auto* p_buffer = p_buffer_t.mutable_data<MT>(ctx.GetPlace());
    auto* g_buffer = p_buffer + LARS_BLOCK_SIZE;

    const int grid_stride = LARS_BLOCK_SIZE * grid_norm;
    const int repeat_times = (numel + grid_stride - 1) / grid_stride - 1;

    L2NormKernel<T, MT><<<grid_norm, LARS_BLOCK_SIZE, 0, cuda_ctx.stream()>>>(
        param_data, grad_data, p_buffer, g_buffer, repeat_times, numel,
        rescale_grad);

    MomentumLarsKernel<
        T, MT><<<grid_lars, LARS_BLOCK_SIZE, 0, cuda_ctx.stream()>>>(
        param_data, grad_data, velocity_data, param_out_data, velocity_out_data,
        master_param_data, master_param_out_data, lr, p_buffer, g_buffer, mu,
        lars_coeff, lars_weight_decay, epsilon, rescale_grad, 0, grid_norm,
        numel);  // 0 is just a placeholder.
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    lars_momentum,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>);
