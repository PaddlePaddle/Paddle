#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/optimizers/lars_momentum_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void MomentumLarsKernel(const T* p, const T* g, const T* v,
                                   const T* learning_rate, const T mu,
                                   const int64_t num, const T lars_coeff,
                                   const T lars_weight_decay, const T* p_norm,
                                   const T* g_norm, T* p_out, T* v_out) {
  T lr = learning_rate[0];
  T local_lr = learning_rate[0];
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    if (p_norm[0] > 0 && g_norm[0] > 0) {
      local_lr = lr * lars_coeff * p_norm[0] /
                 (g_norm[0] + lars_weight_decay * p_norm[0]);
    }
    T v_new = v[i] * mu + local_lr * (g[i] + lars_weight_decay * p[i]);
    v_out[i] = v_new;
    p_out[i] = p[i] - v_new;
  }
}

template <typename DeviceContext, typename T>
class LarsMomentumOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto param_out = ctx.Output<framework::LoDTensor>("ParamOut");
    auto velocity_out = ctx.Output<framework::LoDTensor>("VelocityOut");
    auto param = ctx.Input<framework::LoDTensor>("Param");
    auto velocity = ctx.Input<framework::LoDTensor>("Velocity");
    auto grad = ctx.Input<framework::LoDTensor>("Grad");
    auto learning_rate = ctx.Input<framework::LoDTensor>("LearningRate");

    T* p_out = param_out->mutable_data<T>(ctx.GetPlace());
    T* v_out = velocity_out->mutable_data<T>(ctx.GetPlace());

    T mu = static_cast<T>(ctx.Attr<float>("mu"));
    T lars_coeff = ctx.Attr<float>("lars_coeff");
    T lars_weight_decay = ctx.Attr<float>("lars_weight_decay");

    auto* p = param->data<T>();
    auto* v = velocity->data<T>();
    auto* g = grad->data<T>();
    auto* lr = learning_rate->data<T>();

    int block = 512;
    int grid = (param->numel() + block - 1) / block;

    auto eigen_p = framework::EigenVector<T>::Flatten(*param);
    auto eigen_g = framework::EigenVector<T>::Flatten(*grad);
    // calculate norms using eigein and launch the kernel.
    framework::Tensor p_norm_t, g_norm_t;
    p_norm_t.Resize({1});
    g_norm_t.Resize({1});
    auto* p_norm_data = p_norm_t.mutable_data<T>(ctx.GetPlace());
    auto* g_norm_data = g_norm_t.mutable_data<T>(ctx.GetPlace());
    auto ep_norm = framework::EigenScalar<T>::From(p_norm_t);
    auto eg_norm = framework::EigenScalar<T>::From(g_norm_t);

    auto* place = ctx.template device_context<DeviceContext>().eigen_device();
    ep_norm.device(*place) = eigen_p.square().sum().sqrt();
    eg_norm.device(*place) = eigen_g.square().sum().sqrt();
    MomentumLarsKernel<<<grid, block, 0, ctx.cuda_device_context().stream()>>>(
        p, g, v, lr, mu, param->numel(), lars_coeff, lars_weight_decay,
        p_norm_data, g_norm_data, p_out, v_out);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    lars_momentum,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, double>);
