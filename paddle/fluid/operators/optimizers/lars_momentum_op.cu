#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/operators/optimizers/lars_momentum_op.h"
#include "paddle/fluid/platform/cuda_device_guard.h"
#include "paddle/fluid/platform/fast_divmod.h"

#if defined(__NVCC__) && CUDA_VERSION >= 11000
#include <hip/hip_cooperative_groups.h>
#define LARS_FUNCTION_FLAG __device__
#else
#define LARS_FUNCTION_FLAG __global__
#endif

#ifdef __HIPCC__
#define LARS_BLOCK_SIZE 256
#else
#define LARS_BLOCK_SIZE 512
#endif

#define MAX_MERGED_OPS 200

namespace paddle {
namespace operators {

template <typename T>
using MultiPrecisionType = typename details::MPTypeTrait<T>::Type;

template <typename T, typename MT>
struct MergedParameter {
 public:
  int64_t numel_arr[MAX_MERGED_OPS];
  int repeat_arr[MAX_MERGED_OPS];
  const T* __restrict__ p_arr[MAX_MERGED_OPS];
  const T* __restrict__ g_arr[MAX_MERGED_OPS];
  const MT* __restrict__ v_arr[MAX_MERGED_OPS];
  const MT* __restrict__ lr_arr[MAX_MERGED_OPS];
  const MT* __restrict__ master_p_arr[MAX_MERGED_OPS];
  T* __restrict__ p_out_arr[MAX_MERGED_OPS];
  MT* __restrict__ v_out_arr[MAX_MERGED_OPS];
  MT* __restrict__ master_p_out_arr[MAX_MERGED_OPS];
};

template <typename MT, int VecSize>
__device__ inline void VectorizeLarsUpdate(
    const MT* __restrict__ g, const MT* __restrict__ v, MT* __restrict__ p_out,
    MT* __restrict__ v_out, const MT* __restrict__ p, const MT mu, MT local_lr,
    const MT lars_weight_decay, const MT rescale_grad, const int tid,
    const int grid_stride, const int numel) {
  using VecMType = paddle::platform::AlignedVector<MT, VecSize>;
  int main = numel >> (VecSize >> 1);
  int tail_offset = main * VecSize;

  const VecMType* __restrict__ g_arr = reinterpret_cast<const VecMType*>(g);
  const VecMType* __restrict__ v_arr = reinterpret_cast<const VecMType*>(v);
  const VecMType* __restrict__ p_arr = reinterpret_cast<const VecMType*>(p);
  VecMType* p_out_arr = reinterpret_cast<VecMType*>(p_out);
  VecMType* v_out_arr = reinterpret_cast<VecMType*>(v_out);

  for (int i = tid; i < main; i += grid_stride) {
    VecMType v_new, p_new;
    VecMType g_data = g_arr[i];
    VecMType v_data = v_arr[i];
    VecMType p_data = p_arr[i];

#pragma unroll
    for (int j = 0; j < VecSize; ++j) {
      MT grad = g_data.val[j] * rescale_grad;
      v_new.val[j] =
          fma(v_data.val[j], mu,
              local_lr * fma(lars_weight_decay, p_data.val[j], grad));
      p_new.val[j] = p_data.val[j] - v_new.val[j];
    }
    v_out_arr[i] = v_new;
    p_out_arr[i] = p_new;
  }

  for (int i = tid + tail_offset; i < numel; i += grid_stride) {
    MT grad = g[i] * rescale_grad;
    MT param = p[i];
    MT v_new = fma(v[i], mu, local_lr * fma(lars_weight_decay, param, grad));
    v_out[i] = v_new;
    p_out[i] = param - v_new;
  }
}

template <typename T, typename MT>
__device__ inline void VectorizeLarsUpdateMP(
    const T* __restrict__ g, const MT* __restrict__ v, T* __restrict__ p_out,
    MT* __restrict__ v_out, const MT* __restrict__ master_p,
    MT* __restrict__ master_p_out, const MT mu, MT local_lr,
    const MT lars_weight_decay, const MT rescale_grad, const int tid,
    const int grid_stride, const int numel) {
  // As for multiple-precision, type T and MT cannot be more than fp16 or fp32,
  // Then, the maximum data IO size could be set to 4.
  using VecType = paddle::platform::AlignedVector<T, 4>;
  using VecMType = paddle::platform::AlignedVector<MT, 4>;
  int main = numel >> 2;
  int tail_offset = main << 2;

  const VecType* __restrict__ g_arr = reinterpret_cast<const VecType*>(g);
  const VecMType* __restrict__ v_arr = reinterpret_cast<const VecMType*>(v);
  const VecMType* __restrict__ master_p_arr =
      reinterpret_cast<const VecMType*>(master_p);
  VecType* p_out_arr = reinterpret_cast<VecType*>(p_out);
  VecMType* v_out_arr = reinterpret_cast<VecMType*>(v_out);
  VecMType* master_p_out_arr = reinterpret_cast<VecMType*>(master_p_out);

  for (int i = tid; i < main; i += grid_stride) {
    VecType p_out;
    VecMType v_new, p_new;
    VecType g_data = g_arr[i];
    VecMType v_data = v_arr[i];
    VecMType p_data = master_p_arr[i];

#pragma unroll
    for (int j = 0; j < 4; ++j) {
      MT grad = static_cast<MT>(g_data.val[j]) * rescale_grad;
      v_new.val[j] =
          fma(v_data.val[j], mu,
              local_lr * fma(lars_weight_decay, p_data.val[j], grad));
      p_new.val[j] = p_data.val[j] - v_new.val[j];
      p_out.val[j] = static_cast<T>(p_new.val[j]);
    }
    v_out_arr[i] = v_new;
    p_out_arr[i] = p_out;
    master_p_out_arr[i] = p_new;
  }

  for (int i = tid + tail_offset; i < numel; i += grid_stride) {
    MT grad = static_cast<MT>(g[i]) * rescale_grad;
    MT param = master_p[i];
    MT v_new = fma(v[i], mu, local_lr * fma(lars_weight_decay, param, grad));
    MT p_new = param - v_new;
    v_out[i] = v_new;
    p_out[i] = static_cast<T>(p_new);
    master_p_out[i] = p_new;
  }
}

template <typename T, typename MT>
LARS_FUNCTION_FLAG void L2NormKernel(
    const T* __restrict__ p_data, const T* __restrict__ g_data,
    MT* __restrict__ p_buffer, MT* __restrict__ g_buffer, MT s_buffer[],
    const int64_t numel, const int repeat_times, const MT rescale_grad,
    MT* __restrict__ p_n = nullptr, MT* __restrict__ g_n = nullptr) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int grid_stride = LARS_BLOCK_SIZE * gridDim.x;
  const MT rescale_grad_pow = rescale_grad * rescale_grad;
  MT p_arr_val = static_cast<MT>(0);
  MT g_arr_val = static_cast<MT>(0);

  if (repeat_times == 0) {
    if (tid < numel) {
      p_arr_val = static_cast<MT>(p_data[tid]);
      g_arr_val = static_cast<MT>(g_data[tid]);
    }
    s_buffer[0] += math::blockReduceSum<MT>(p_arr_val * p_arr_val, FINAL_MASK);
    s_buffer[1] += math::blockReduceSum<MT>(g_arr_val * g_arr_val, FINAL_MASK);
  } else {
    /* To avoid occupy too much temp buffer. Hence, slice the whole data into 2
    parts, the front of them whose quantity is excatly multiple of grid-thread
    number, and this part of data is delt in for loop, the rest of data is delt
    with another step to avoid visiting data address beyond bound. */
    for (int i = 0; i < repeat_times; ++i) {
      p_arr_val = static_cast<MT>(p_data[tid]);
      g_arr_val = static_cast<MT>(g_data[tid]);
      tid += grid_stride;
      s_buffer[0] +=
          math::blockReduceSum<MT>(p_arr_val * p_arr_val, FINAL_MASK);
      s_buffer[1] +=
          math::blockReduceSum<MT>(g_arr_val * g_arr_val, FINAL_MASK);
      __syncthreads();
    }
    MT p_val = 0;
    MT g_val = 0;
    if (tid < numel) {
      p_val = static_cast<MT>(p_data[tid]);
      g_val = static_cast<MT>(g_data[tid]);
    }
    s_buffer[0] += math::blockReduceSum<MT>(p_val * p_val, FINAL_MASK);
    s_buffer[1] += math::blockReduceSum<MT>(g_val * g_val, FINAL_MASK);
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    p_buffer[blockIdx.x] = s_buffer[0];
    g_buffer[blockIdx.x] = rescale_grad_pow * s_buffer[1];
  }
  // Grid sync for completely writring partial result back to gloabl memory
  const cooperative_groups::grid_group cg = cooperative_groups::this_grid();
  cg.sync();
  MT p_partial_sum = threadIdx.x < gridDim.x ? p_buffer[threadIdx.x] : 0;
  MT g_partial_sum = threadIdx.x < gridDim.x ? g_buffer[threadIdx.x] : 0;
  *p_n = sqrt(math::blockReduceSum<MT>(p_partial_sum, FINAL_MASK));
  *g_n = sqrt(math::blockReduceSum<MT>(g_partial_sum, FINAL_MASK));
}

template <typename T, typename MT>
__global__ void MergedMomentumLarsKernel(
    MergedParameter<T, MT>* merged_param, MT* __restrict__ p_buffer,
    MT* __restrict__ g_buffer, const int op_num, const MT mu,
    const MT lars_coeff, const MT lars_weight_decay, const MT epsilon,
    const MT rescale_grad, const bool use_master_data) {
  __shared__ MT s_buffer[2];
  int grid_stride = gridDim.x * LARS_BLOCK_SIZE;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = 0; i < op_num; ++i) {
    int numel = merged_param->numel_arr[i];
    MT p_n = static_cast<MT>(0);
    MT g_n = static_cast<MT>(0);
    s_buffer[0] = static_cast<MT>(0);
    s_buffer[1] = static_cast<MT>(0);
    L2NormKernel<T, MT>(merged_param->p_arr[i], merged_param->g_arr[i],
                        p_buffer, g_buffer, s_buffer, numel,
                        merged_param->repeat_arr[i], rescale_grad, &p_n, &g_n);
    const MT lr = *(merged_param->lr_arr[i]);
    MT local_lr = lr;
    if (lars_weight_decay > static_cast<MT>(0)) {
      local_lr =
          lr * lars_coeff * p_n / (fma(lars_weight_decay, p_n, g_n) + epsilon);
    }

    if (use_master_data) {
      VectorizeLarsUpdateMP<T, MT>(
          merged_param->g_arr[i], merged_param->v_arr[i],
          merged_param->p_out_arr[i], merged_param->v_out_arr[i],
          merged_param->master_p_arr[i], merged_param->master_p_out_arr[i], mu,
          local_lr, lars_weight_decay, rescale_grad, tid, grid_stride, numel);
    } else {
      if (std::is_same<T, float>::value ||
          std::is_same<T, paddle::platform::float16>::value) {
        VectorizeLarsUpdate<MT, 4>(
            reinterpret_cast<const MT*>(merged_param->g_arr[i]),
            merged_param->v_arr[i],
            reinterpret_cast<MT*>(merged_param->p_out_arr[i]),
            merged_param->v_out_arr[i],
            reinterpret_cast<const MT*>(merged_param->p_arr[i]), mu, local_lr,
            lars_weight_decay, rescale_grad, tid, grid_stride, numel);
      } else {
        VectorizeLarsUpdate<MT, 2>(
            reinterpret_cast<const MT*>(merged_param->g_arr[i]),
            merged_param->v_arr[i],
            reinterpret_cast<MT*>(merged_param->p_out_arr[i]),
            merged_param->v_out_arr[i],
            reinterpret_cast<const MT*>(merged_param->p_arr[i]), mu, local_lr,
            lars_weight_decay, rescale_grad, tid, grid_stride, numel);
      }
    }
  }
}

template <typename T, typename MT>
__global__ void MomentumLarsKernel(
    const T* __restrict__ p, const T* __restrict__ g, const MT* __restrict__ v,
    T* p_out, MT* v_out, const MT* __restrict__ master_p,
    MT* __restrict__ master_p_out, const MT* __restrict__ learning_rate,
    MT* __restrict__ p_buffer, MT* __restrict__ g_buffer, const MT mu,
    const MT lars_coeff, const MT lars_weight_decay, const MT epsilon,
    const MT rescale_grad, const int repeat_times, const int thresh,
    const int64_t numel) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int grid_stride = gridDim.x * LARS_BLOCK_SIZE;
  __shared__ MT s_buffer[2];
  s_buffer[0] = static_cast<MT>(0);
  s_buffer[1] = static_cast<MT>(0);
  MT p_n = static_cast<MT>(0);
  MT g_n = static_cast<MT>(0);
  L2NormKernel<T, MT>(p, g, p_buffer, g_buffer, s_buffer, numel, repeat_times,
                      rescale_grad, &p_n, &g_n);
  const MT lr = learning_rate[0];
  MT local_lr = lr;
  if (lars_weight_decay > static_cast<MT>(0)) {
    local_lr =
        lr * lars_coeff * p_n / (fma(lars_weight_decay, p_n, g_n) + epsilon);
  }

  if (master_p) {
    VectorizeLarsUpdateMP<T, MT>(g, v, p_out, v_out, master_p, master_p_out, mu,
                                 local_lr, lars_weight_decay, rescale_grad, tid,
                                 grid_stride, numel);
  } else {
    if (std::is_same<T, float>::value ||
        std::is_same<T, paddle::platform::float16>::value) {
      VectorizeLarsUpdate<MT, 4>(
          reinterpret_cast<const MT*>(g), v, reinterpret_cast<MT*>(p_out),
          v_out, reinterpret_cast<const MT*>(p), mu, local_lr,
          lars_weight_decay, rescale_grad, tid, grid_stride, numel);
    } else {
      VectorizeLarsUpdate<MT, 2>(
          reinterpret_cast<const MT*>(g), v, reinterpret_cast<MT*>(p_out),
          v_out, reinterpret_cast<const MT*>(p), mu, local_lr,
          lars_weight_decay, rescale_grad, tid, grid_stride, numel);
    }
  }
}

template <typename DeviceContext, typename T>
class LarsMomentumOpCUDAKernel : public framework::OpKernel<T> {
  using MT = MultiPrecisionType<T>;

 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    int op_num = 1;
    bool multi_precision = ctx.Attr<bool>("multi_precision");
    const bool merge_operation = ctx.Attr<bool>("merge_operation");
    bool has_master = false;
    int num_blocks_per_sm = 0;
    auto& cuda_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    int sm_num = cuda_ctx.GetSMCount();
    framework::Tensor tmp_buffer_t =
        ctx.AllocateTmpTensor<MT, platform::CUDADeviceContext>(
            {LARS_BLOCK_SIZE << 1}, cuda_ctx);
    auto* p_buffer = tmp_buffer_t.mutable_data<MT>(ctx.GetPlace());
    auto* g_buffer = p_buffer + LARS_BLOCK_SIZE;

    MT mu = static_cast<MT>(ctx.Attr<float>("mu"));
    MT lars_coeff = static_cast<MT>(ctx.Attr<float>("lars_coeff"));
    MT lars_weight_decay =
        static_cast<MT>(ctx.Attr<float>("lars_weight_decay"));
    MT epsilon = static_cast<MT>(ctx.Attr<float>("epsilon"));
    MT rescale_grad = static_cast<MT>(ctx.Attr<float>("rescale_grad"));

    if (merge_operation) {
      auto grad = ctx.MultiInput<framework::LoDTensor>("Grad");
      op_num = grad.size();
    }
    PADDLE_ENFORCE_GT(
        op_num, MAX_MERGED_OPS,
        platform::errors::InvalidArgument(
            "Currently, the maximum quantity of merged-op supported is (%d), "
            "but lars op required for trainning this model is (%d)\n",
            MAX_MERGED_OPS, op_num));

    int max_numel = 0;
    MergedParameter<T, MT> merged_params;

    if (merge_operation) {
      auto param = ctx.MultiInput<framework::LoDTensor>("Param");
      auto velocity = ctx.MultiInput<framework::LoDTensor>("Velocity");
      auto grad = ctx.MultiInput<framework::LoDTensor>("Grad");
      auto learning_rate = ctx.MultiInput<framework::LoDTensor>("LearningRate");
      auto param_out = ctx.MultiOutput<framework::LoDTensor>("ParamOut");
      auto velocity_out = ctx.MultiOutput<framework::LoDTensor>("VelocityOut");
      for (int i = 0; i < op_num; ++i) {
        int temp_numel = param[i]->numel();
        max_numel = max_numel < temp_numel ? temp_numel : max_numel;
        merged_params.numel_arr[i] = temp_numel;
        merged_params.p_arr[i] = param[i]->data<T>();
        merged_params.g_arr[i] = grad[i]->data<T>();
        merged_params.v_arr[i] = velocity[i]->data<MT>();
        merged_params.lr_arr[i] = learning_rate[i]->data<MT>();
        merged_params.p_out_arr[i] =
            param_out[i]->mutable_data<T>(ctx.GetPlace());
        merged_params.v_out_arr[i] =
            velocity_out[i]->mutable_data<MT>(ctx.GetPlace());
      }
      int grid = (max_numel + LARS_BLOCK_SIZE - 1) / LARS_BLOCK_SIZE;
      hipOccupancyMaxActiveBlocksPerMultiprocessor(
          &num_blocks_per_sm, MergedMomentumLarsKernel<T, MT>, LARS_BLOCK_SIZE,
          sizeof(MT) << 1);
      int grid_real =
          std::min(std::min(sm_num * num_blocks_per_sm, grid), LARS_BLOCK_SIZE);
      int grid_stride = LARS_BLOCK_SIZE * grid_real;

      for (int i = 0; i < op_num; ++i) {
        merged_params.repeat_arr[i] =
            (merged_params.numel_arr[i] + grid_stride - 1) / grid_stride - 1;
      }
      if (multi_precision) {
        auto master_param = ctx.MultiInput<framework::LoDTensor>("MasterParam");
        auto master_param_out =
            ctx.MultiOutput<framework::LoDTensor>("MasterParamOut");
        for (int i = 0; i < op_num; ++i) {
          merged_params.master_p_arr[i] = master_param[i]->data<MT>();
          merged_params.master_p_out_arr[i] =
              master_param_out[i]->mutable_data<MT>(ctx.GetPlace());
        }
      }
      auto merged_buf = memory::Alloc(cuda_ctx, sizeof(merged_params));
      auto* merged_ptr =
          reinterpret_cast<MergedParameter<T, MT>*>(merged_buf->ptr());
      memory::Copy(BOOST_GET_CONST(platform::CUDAPlace, cuda_ctx.GetPlace()),
                   reinterpret_cast<void*>(merged_ptr), platform::CPUPlace(),
                   reinterpret_cast<void*>(&merged_params),
                   sizeof(merged_params), cuda_ctx.stream());
      void* cuda_param[] = {reinterpret_cast<void*>(&merged_ptr),
                            reinterpret_cast<void*>(&p_buffer),
                            reinterpret_cast<void*>(&g_buffer),
                            reinterpret_cast<void*>(&op_num),
                            reinterpret_cast<void*>(&mu),
                            reinterpret_cast<void*>(&lars_coeff),
                            reinterpret_cast<void*>(&lars_weight_decay),
                            reinterpret_cast<void*>(&epsilon),
                            reinterpret_cast<void*>(&rescale_grad),
                            reinterpret_cast<void*>(&multi_precision)};
      // Lanuch all sm theads.
      hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
          reinterpret_cast<void*>(MomentumLarsKernel<T), MT>), grid_real,
          LARS_BLOCK_SIZE, cuda_param, 0, cuda_ctx.stream());
    } else {
      auto param = ctx.Input<framework::LoDTensor>("Param");
      auto grad = ctx.Input<framework::LoDTensor>("Grad");
      auto velocity = ctx.Input<framework::LoDTensor>("Velocity");
      auto learning_rate = ctx.Input<framework::LoDTensor>("LearningRate");
      auto param_out = ctx.Output<framework::LoDTensor>("ParamOut");
      auto velocity_out = ctx.Output<framework::LoDTensor>("VelocityOut");

      auto* p = param->data<T>();
      auto* g = grad->data<T>();
      auto* v = velocity->data<MT>();
      auto* lr = learning_rate->data<MT>();
      auto* p_out = param_out->mutable_data<T>(ctx.GetPlace());
      auto* v_out = velocity_out->mutable_data<MT>(ctx.GetPlace());
      const MT* master_p = nullptr;
      MT* master_p_out = nullptr;
      if (multi_precision) {
        auto master_param = ctx.Input<framework::Tensor>("MasterParam");
        auto master_param_out = ctx.Output<framework::Tensor>("MasterParamOut");
        master_p = master_param->data<MT>();
        master_p_out = master_param_out->mutable_data<MT>(ctx.GetPlace());
      }
      int64_t numel = param->numel();
      hipOccupancyMaxActiveBlocksPerMultiprocessor(
          &num_blocks_per_sm, MomentumLarsKernel<T, MT>, LARS_BLOCK_SIZE,
          sizeof(MT) << 1);
      int grid = (numel + LARS_BLOCK_SIZE - 1) / LARS_BLOCK_SIZE;
      int grid_real =
          std::min(std::min(sm_num * num_blocks_per_sm, grid), LARS_BLOCK_SIZE);
      int grid_stride = LARS_BLOCK_SIZE * grid_real;
      int repeat_times = (numel + grid_stride - 1) / grid_stride - 1;
      int thresh = 0;

      // Uniform kernel parameter for hipLaunchCooperativeKernel
      void* cuda_param[] = {
          reinterpret_cast<void*>(&p),
          reinterpret_cast<void*>(&g),
          reinterpret_cast<void*>(&v),
          reinterpret_cast<void*>(&p_out),
          reinterpret_cast<void*>(&v_out),
          reinterpret_cast<void*>(&master_p),
          reinterpret_cast<void*>(&master_p_out),
          reinterpret_cast<void*>(&lr),
          reinterpret_cast<void*>(&p_buffer),
          reinterpret_cast<void*>(&g_buffer),
          reinterpret_cast<void*>(&mu),
          reinterpret_cast<void*>(&lars_coeff),
          reinterpret_cast<void*>(&lars_weight_decay),
          reinterpret_cast<void*>(&epsilon),
          reinterpret_cast<void*>(&rescale_grad),
          reinterpret_cast<void*>(&repeat_times),
          reinterpret_cast<void*>(&thresh),  // Just a placeholder
          reinterpret_cast<void*>(&numel)};
      // Lanuch all sm theads.
      hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
          reinterpret_cast<void*>(MomentumLarsKernel<T), MT>), grid_real,
          LARS_BLOCK_SIZE, cuda_param, 0, cuda_ctx.stream());
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    lars_momentum,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>);
