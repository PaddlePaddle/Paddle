#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/optimizers/lars_momentum_op.h"
#include "paddle/fluid/platform/fast_divmod.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"

#if CUDA_VERSION >= 11000
#include <hip/hip_cooperative_groups.h>
#endif

#ifdef __HIPCC__
#define LARS_BLOCK_SIZE 256
#else
#define LARS_BLOCK_SIZE 512
#endif

#define LARS_MAX_MERGED_OPS 60

namespace paddle {
namespace operators {

template <typename T>
using MultiPrecisionType = typename details::MPTypeTrait<T>::Type;

__device__ __forceinline__ float Sqrt(float x) { return sqrtf(x); }
__device__ __forceinline__ double Sqrt(double x) { return sqrt(x); }
__device__ __forceinline__ float Fma(float x, float y, float z) {
  return fmaf(x, y, z);
}
__device__ __forceinline__ double Fma(double x, double y, double z) {
  return fma(x, y, z);
}

template <typename T>
class LarsThreadConfig {
 public:
  int grid_for_norm;
  int grid_for_lars;
#if CUDA_VERSION >= 11000

 private:
  int grid_stride;

 public:
  explicit LarsThreadConfig(int64_t numel, int sm_num, int num_blocks_per_sm) {
    int grid = (numel + LARS_BLOCK_SIZE - 1) / LARS_BLOCK_SIZE;
    grid_for_lars =
        std::min(std::min(sm_num * num_blocks_per_sm, grid), LARS_BLOCK_SIZE);
    grid_stride = LARS_BLOCK_SIZE * grid_for_lars;
  }

  int GetRepeatTimes(int64_t numel) {
    return (numel + grid_stride - 1) / grid_stride - 1;
  }
#else
  int repeat_times;
  explicit LarsThreadConfig(const int64_t numel) {
    int grid = (numel + LARS_BLOCK_SIZE - 1) / LARS_BLOCK_SIZE;
    grid_for_norm = std::min(grid, LARS_BLOCK_SIZE);
    const int grid_stride = grid_for_norm * LARS_BLOCK_SIZE;
    repeat_times = (numel + grid_stride - 1) / grid_stride - 1;
    // Determine to read 4 fp16 or float data once, but 2 double data once.
    grid_for_lars =
        std::is_same<double, T>::value
            ? (numel + (LARS_BLOCK_SIZE << 1) - 1) / (LARS_BLOCK_SIZE << 1)
            : (numel + (LARS_BLOCK_SIZE << 2) - 1) / (LARS_BLOCK_SIZE << 2);
  }
#endif
};

template <typename T, typename MT, int VecSize, bool IsAmp = false>
__device__ inline void VectorizeLarsUpdate(
    const T* __restrict__ grad, const MT* param, const MT* velocity,
    T* param_out, MT* velocity_out, const MT mu, MT local_lr,
    const MT lars_weight_decay, const MT rescale_grad, const int tid,
    const int grid_stride, const int numel, MT* master_param_out = nullptr) {
  using VecType = paddle::platform::AlignedVector<T, VecSize>;
  using VecMType = paddle::platform::AlignedVector<MT, VecSize>;
  int main = numel >> (VecSize >> 1);
  int tail_offset = main * VecSize;

  const VecType* grad_vec = reinterpret_cast<const VecType*>(grad);
  const VecMType* param_vec = reinterpret_cast<const VecMType*>(param);
  const VecMType* velocity_vec = reinterpret_cast<const VecMType*>(velocity);
  VecType* param_out_vec = reinterpret_cast<VecType*>(param_out);
  VecMType* velocity_out_vec = reinterpret_cast<VecMType*>(velocity_out);

  VecMType* master_param_out_vec;
  if (IsAmp) {
    master_param_out_vec = reinterpret_cast<VecMType*>(master_param_out);
  }

  for (int i = tid; i < main; i += grid_stride) {
    VecType param_out_tmp;
    VecMType velocity_tmp, param_tmp;
    VecType grad_data = grad_vec[i];
    VecMType param_data = param_vec[i];
    VecMType velocity_data = velocity_vec[i];
#pragma unroll
    for (int j = 0; j < VecSize; ++j) {
      MT grad_val = static_cast<MT>(grad_data[j]) * rescale_grad;
      velocity_tmp[j] =
          Fma(velocity_data[j], mu,
              local_lr * Fma(lars_weight_decay, param_data[j], grad_val));
      param_tmp[j] = param_data[j] - velocity_tmp[j];
      param_out_tmp[j] = static_cast<T>(param_tmp[j]);
    }
    param_out_vec[i] = param_out_tmp;
    velocity_out_vec[i] = velocity_tmp;
    if (IsAmp) {
      master_param_out_vec[i] = param_tmp;
    }
  }

  for (int i = tid + tail_offset; i < numel; i += grid_stride) {
    MT grad_val = static_cast<MT>(grad[i]) * rescale_grad;
    MT param_val = param[i];
    MT velocity_tmp = Fma(velocity[i], mu, local_lr * Fma(lars_weight_decay,
                                                          param_val, grad_val));
    MT param_tmp = param_val - velocity_tmp;
    param_out[i] = static_cast<T>(param_tmp);
    velocity_out[i] = velocity_tmp;
    if (IsAmp) {
      master_param_out[i] = param_tmp;
    }
  }
}

#if CUDA_VERSION >= 11000
/* Once CUDA_VERSION is beyond 11, cooperative_groups can be involved in without
  --rdc=true compile flag, then L2_norm kernel can be set with __device__ and
  cooperative_groups::grid_group also can be involved. Otherwise, adding this
  flag may affect much, L2_norm kernel shall be set with __global__.*/
// TODO(limingshu): declaration of cooperative_groups wapper is invalid in host.
template <typename T, typename MT>
__forceinline__ __device__ void L2NormKernel(
    const cooperative_groups::grid_group* cg,
#else
template <typename T, typename MT>
__global__ void L2NormKernel(
#endif
    const T* p_data, const T* __restrict__ g_data, MT* __restrict__ p_buffer,
    MT* __restrict__ g_buffer, const int64_t numel, const int repeat_times,
    const MT rescale_grad, const int thresh = 0, MT* __restrict__ p_n = nullptr,
    MT* __restrict__ g_n = nullptr) {
  __shared__ MT s_buffer[2];
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int grid_stride = LARS_BLOCK_SIZE * gridDim.x;

  MT p_tmp = static_cast<MT>(0);
  MT g_tmp = static_cast<MT>(0);
  while (tid < numel) {
    MT tmp0 = static_cast<MT>(p_data[tid]);
    MT tmp1 = static_cast<MT>(g_data[tid]);
    p_tmp += (tmp0 * tmp0);
    g_tmp += (tmp1 * tmp1);
    tid += grid_stride;
  }
  p_tmp = phi::funcs::blockReduceSum<MT>(p_tmp, FINAL_MASK);
  g_tmp = phi::funcs::blockReduceSum<MT>(g_tmp, FINAL_MASK);

  if (threadIdx.x == 0) {
    p_buffer[blockIdx.x] = p_tmp;
    g_buffer[blockIdx.x] = g_tmp;
  }
#if CUDA_VERSION >= 11000
  cg->sync();  // Grid sync for writring partial result to gloabl memory
  MT p_part_sum = threadIdx.x < gridDim.x ? p_buffer[threadIdx.x] : 0;
  MT g_part_sum = threadIdx.x < gridDim.x ? g_buffer[threadIdx.x] : 0;
  MT tmp0 = phi::funcs::blockReduceSum<MT>(p_part_sum, FINAL_MASK);
  MT tmp1 = phi::funcs::blockReduceSum<MT>(g_part_sum, FINAL_MASK);
  if (threadIdx.x == 0) {
    s_buffer[0] = tmp0;
    s_buffer[1] = tmp1;
  }
  __syncthreads();
  *p_n = Sqrt(s_buffer[0]);
  *g_n = rescale_grad * Sqrt(s_buffer[1]);
#endif
}

template <typename T, typename MT>
__forceinline__ __device__ void MomentumUpdate(
    const T* param, const T* __restrict__ grad, const MT* velocity,
    T* param_out, MT* velocity_out, const MT* master_param,
    MT* master_param_out, const MT* __restrict__ learning_rate, const MT mu,
    const MT lars_weight_decay, const MT lars_coeff, const MT epsilon,
    const MT rescale_grad, const MT param_norm, const MT grad_norm,
    const int tid, const int grid_stride, const int64_t numel,
    const bool is_amp) {
  const MT lr = learning_rate[0];
  MT local_lr = lr;
  if (lars_weight_decay > static_cast<MT>(0)) {
    local_lr = lr * lars_coeff * param_norm /
               (fma(lars_weight_decay, param_norm, grad_norm) + epsilon);
  }
  if (is_amp) {
    VectorizeLarsUpdate<T, MT, /*VecSize=*/4, /*IsAmp=*/true>(
        grad, master_param, velocity, param_out, velocity_out, mu, local_lr,
        lars_weight_decay, rescale_grad, tid, grid_stride, numel,
        master_param_out);
  } else {
    if (std::is_same<T, float>::value ||
        std::is_same<T, paddle::platform::float16>::value) {
      /* TODO(limingshu): pointer cast may damage memory accessing for fp16 */
      VectorizeLarsUpdate<T, MT, /*VecSize=*/4, /*IsAmp=*/false>(
          grad, reinterpret_cast<const MT*>(param), velocity, param_out,
          velocity_out, mu, local_lr, lars_weight_decay, rescale_grad, tid,
          grid_stride, numel);
    } else {
      VectorizeLarsUpdate<T, MT, /*VecSize=*/2, /*IsAmp=*/false>(
          grad, reinterpret_cast<const MT*>(param), velocity, param_out,
          velocity_out, mu, local_lr, lars_weight_decay, rescale_grad, tid,
          grid_stride, numel);
    }
  }
}

#if CUDA_VERSION >= 11000
template <typename T, typename MT>
struct LarsParamWarpper {
  int64_t numel_arr[LARS_MAX_MERGED_OPS];
  int repeat_arr[LARS_MAX_MERGED_OPS];
  const T* __restrict__ g_arr[LARS_MAX_MERGED_OPS];
  const MT* __restrict__ lr_arr[LARS_MAX_MERGED_OPS];
  T* __restrict__ p_out_arr[LARS_MAX_MERGED_OPS];
  MT* __restrict__ v_out_arr[LARS_MAX_MERGED_OPS];
  MT* __restrict__ master_p_out_arr[LARS_MAX_MERGED_OPS];
  MT weight_decay_arr[LARS_MAX_MERGED_OPS];
};

template <typename T, typename MT>
__global__ void MergedMomentumLarsKernel(LarsParamWarpper<T, MT> lars_warpper,
                                         MT* __restrict__ p_buffer,
                                         MT* __restrict__ g_buffer,
                                         const int op_num, const MT mu,
                                         const MT lars_coeff, const MT epsilon,
                                         const MT rescale_grad,
                                         const bool is_amp) {
  int grid_stride = gridDim.x * LARS_BLOCK_SIZE;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  const cooperative_groups::grid_group cg = cooperative_groups::this_grid();
  for (int i = 0; i < op_num; ++i) {
    int numel = lars_warpper.numel_arr[i];
    MT param_norm = static_cast<MT>(0);
    MT grad_norm = static_cast<MT>(0);
    L2NormKernel<T, MT>(&cg, lars_warpper.p_out_arr[i], lars_warpper.g_arr[i],
                        p_buffer, g_buffer, numel, lars_warpper.repeat_arr[i],
                        rescale_grad, 0, &param_norm, &grad_norm);
    MomentumUpdate<T, MT>(
        lars_warpper.p_out_arr[i], lars_warpper.g_arr[i],
        lars_warpper.v_out_arr[i], lars_warpper.p_out_arr[i],
        lars_warpper.v_out_arr[i], lars_warpper.master_p_out_arr[i],
        lars_warpper.master_p_out_arr[i], lars_warpper.lr_arr[i], mu,
        lars_warpper.weight_decay_arr[i], lars_coeff, epsilon, rescale_grad,
        param_norm, grad_norm, tid, grid_stride, numel, is_amp);
  }
}
#endif

template <typename T, typename MT>
__global__ void MomentumLarsKernel(
    const T* param, const T* __restrict__ grad, const MT* velocity,
    T* param_out, MT* velocity_out, const MT* master_param,
    MT* master_param_out, const MT* __restrict__ learning_rate,
    MT* __restrict__ p_buffer, MT* __restrict__ g_buffer, const MT mu,
    const MT lars_coeff, const MT lars_weight_decay, const MT epsilon,
    const MT rescale_grad, const int repeat_times, const int thresh,
    const int64_t numel, const bool is_amp) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int grid_stride = gridDim.x * LARS_BLOCK_SIZE;
#if CUDA_VERSION >= 11000
  const cooperative_groups::grid_group cg = cooperative_groups::this_grid();
  MT param_norm = static_cast<MT>(0);
  MT grad_norm = static_cast<MT>(0);
  L2NormKernel<T, MT>(&cg, param, grad, p_buffer, g_buffer, numel, repeat_times,
                      rescale_grad, gridDim.x, &param_norm, &grad_norm);
#else
  const MT rescale_grad_pow = rescale_grad * rescale_grad;
  MT param_part_norm = threadIdx.x < thresh ? p_buffer[threadIdx.x] : 0;
  MT grad_part_norm = threadIdx.x < thresh ? g_buffer[threadIdx.x] : 0;
  __syncthreads();
  MT param_norm =
      Sqrt(phi::funcs::blockReduceSum<MT>(param_part_norm, FINAL_MASK));
  MT grad_norm = Sqrt(rescale_grad_pow * phi::funcs::blockReduceSum<MT>(
                                             grad_part_norm, FINAL_MASK));
#endif
  MomentumUpdate<T, MT>(param, grad, velocity, param_out, velocity_out,
                        master_param, master_param_out, learning_rate, mu,
                        lars_weight_decay, lars_coeff, epsilon, rescale_grad,
                        param_norm, grad_norm, tid, grid_stride, numel, is_amp);
}

template <typename T, typename MT>
inline void SeparatedLarsMomentumOpCUDAKernel(
    const platform::CUDADeviceContext& cuda_ctx, const T* param_data,
    T* param_out_data, const MT* velocity_data, MT* velocity_out_data,
    const T* grad_data, const MT* lr, MT* p_buffer, MT* g_buffer, const MT mu,
    const MT lars_coeff, const MT weight_decay, const MT epsilon,
    const MT rescale_grad, const int64_t numel, const MT* master_param_data,
    MT* master_out_data, const bool is_amp) {
  LarsThreadConfig<T> lars_thread_config(numel);
  L2NormKernel<T, MT><<<lars_thread_config.grid_for_norm, LARS_BLOCK_SIZE, 0,
                        cuda_ctx.stream()>>>(
      param_data, grad_data, p_buffer, g_buffer, numel,
      lars_thread_config.repeat_times, rescale_grad);

  MomentumLarsKernel<T, MT><<<lars_thread_config.grid_for_lars, LARS_BLOCK_SIZE,
                              0, cuda_ctx.stream()>>>(
      param_data, grad_data, velocity_data, param_out_data, velocity_out_data,
      master_param_data, master_out_data, lr, p_buffer, g_buffer, mu,
      lars_coeff, weight_decay, epsilon, rescale_grad, 0,
      lars_thread_config.grid_for_norm, numel, is_amp);
}

template <typename DeviceContext, typename T>
class LarsMomentumOpCUDAKernel : public framework::OpKernel<T> {
  using MT = MultiPrecisionType<T>;

 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    int num_blocks_per_sm = 0;
    bool multi_precision = ctx.Attr<bool>("multi_precision");
    auto& cuda_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    int sm_num = cuda_ctx.GetSMCount();
    framework::Tensor tmp_buffer_t =
        ctx.AllocateTmpTensor<MT, platform::CUDADeviceContext>(
            {LARS_BLOCK_SIZE << 1}, cuda_ctx);
    auto* p_buffer = tmp_buffer_t.mutable_data<MT>(ctx.GetPlace());
    auto* g_buffer = p_buffer + LARS_BLOCK_SIZE;

    MT mu = static_cast<MT>(ctx.Attr<float>("mu"));
    MT lars_coeff = static_cast<MT>(ctx.Attr<float>("lars_coeff"));
    MT epsilon = static_cast<MT>(ctx.Attr<float>("epsilon"));
    MT rescale_grad = static_cast<MT>(ctx.Attr<float>("rescale_grad"));

    auto weight_decay_arr = ctx.Attr<std::vector<float>>("lars_weight_decay");
    auto grad = ctx.MultiInput<framework::LoDTensor>("Grad");
    auto param = ctx.MultiInput<framework::LoDTensor>("Param");
    auto velocity = ctx.MultiInput<framework::LoDTensor>("Velocity");
    auto param_out = ctx.MultiOutput<framework::LoDTensor>("ParamOut");
    auto velocity_out = ctx.MultiOutput<framework::LoDTensor>("VelocityOut");
    auto learning_rate = ctx.MultiInput<framework::LoDTensor>("LearningRate");
    auto master_param = ctx.MultiInput<framework::LoDTensor>("MasterParam");
    auto master_param_out =
        ctx.MultiOutput<framework::LoDTensor>("MasterParamOut");

    int op_num = grad.size();
#if CUDA_VERSION >= 11000
    if (op_num > 1) {
      LarsParamWarpper<T, MT> lars_warpper;
      PADDLE_ENFORCE_LT(
          op_num, LARS_MAX_MERGED_OPS,
          platform::errors::InvalidArgument(
              "The maximum number of merged-ops supported is (%d), but"
              "lars op required for trainning this model is (%d)\n",
              LARS_MAX_MERGED_OPS, op_num));

      /* Implementation of lars optimizer consists of following two steps:
        1. Figure out the L2 norm statistic result of grad data and param data.
        2. Update param and velocity with usage of L2 norm statistic result.
      Step1 and step2 can be merged with api provided by nvida
        hipLaunchCooperativeKernel:
        - The thread quantity shall less than pyhsical SM limited threads
        - Launche as thread-block can synchronizlly execute. */
      hipOccupancyMaxActiveBlocksPerMultiprocessor(
          &num_blocks_per_sm, MergedMomentumLarsKernel<T, MT>, LARS_BLOCK_SIZE,
          sizeof(MT) << 1);

      size_t total_numel = 0;
      for (int i = 0; i < op_num; ++i) {
        size_t temp_numel = param[i]->numel();
        total_numel += temp_numel;
        lars_warpper.numel_arr[i] = temp_numel;
        lars_warpper.g_arr[i] = grad[i]->data<T>();
        lars_warpper.lr_arr[i] = learning_rate[i]->data<MT>();
        lars_warpper.p_out_arr[i] =
            param_out[i]->mutable_data<T>(ctx.GetPlace());
        lars_warpper.v_out_arr[i] =
            velocity_out[i]->mutable_data<MT>(ctx.GetPlace());
        lars_warpper.weight_decay_arr[i] = static_cast<MT>(weight_decay_arr[i]);
        PADDLE_ENFORCE_EQ(
            param[i]->data<T>(), lars_warpper.p_out_arr[i],
            platform::errors::InvalidArgument(
                "Input(Param) and Output(ParamOut) must be the same Tensors."));
        PADDLE_ENFORCE_EQ(velocity[i]->data<MT>(), lars_warpper.v_out_arr[i],
                          platform::errors::InvalidArgument(
                              "Input(Velocity) and Output(VelocityOut) must be "
                              "the same Tensors."));
      }
      int64_t avg_numel = total_numel / op_num;
      LarsThreadConfig<float> lars_thread_config(avg_numel, sm_num,
                                                 num_blocks_per_sm);
      for (int i = 0; i < op_num; ++i) {
        lars_warpper.repeat_arr[i] =
            lars_thread_config.GetRepeatTimes(lars_warpper.numel_arr[i]);
      }
      if (multi_precision) {
        for (int i = 0; i < op_num; ++i) {
          lars_warpper.master_p_out_arr[i] =
              master_param_out[i]->mutable_data<MT>(ctx.GetPlace());
          PADDLE_ENFORCE_EQ(master_param[i]->data<MT>(),
                            lars_warpper.master_p_out_arr[i],
                            platform::errors::InvalidArgument(
                                "Input(MasterParam) and Output(MasterParamOut) "
                                "must be the same Tensors."));
        }
      }
      void* cuda_param[] = {reinterpret_cast<void*>(&lars_warpper),
                            reinterpret_cast<void*>(&p_buffer),
                            reinterpret_cast<void*>(&g_buffer),
                            reinterpret_cast<void*>(&op_num),
                            reinterpret_cast<void*>(&mu),
                            reinterpret_cast<void*>(&lars_coeff),
                            reinterpret_cast<void*>(&epsilon),
                            reinterpret_cast<void*>(&rescale_grad),
                            reinterpret_cast<void*>(&multi_precision)};
      // Lanuch all sm theads, and thead of each block synchronizedly cooperate.
      hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
          reinterpret_cast<void*>(MergedMomentumLarsKernel<T), MT>),
          lars_thread_config.grid_for_lars, LARS_BLOCK_SIZE, cuda_param, 0,
          cuda_ctx.stream());
    } else {
      auto* param_data = param[0]->data<T>();
      auto* grad_data = grad[0]->data<T>();
      auto* velocity_data = velocity[0]->data<MT>();
      auto* lr = learning_rate[0]->data<MT>();
      auto* param_out_data = param_out[0]->mutable_data<T>(ctx.GetPlace());
      auto* velocity_out_data =
          velocity_out[0]->mutable_data<MT>(ctx.GetPlace());
      const MT* master_param_data =
          multi_precision ? master_param[0]->data<MT>() : nullptr;
      MT* master_param_out_data =
          multi_precision
              ? master_param_out[0]->mutable_data<MT>(ctx.GetPlace())
              : nullptr;
      int64_t numel = param[0]->numel();
      MT lars_weight_decay = weight_decay_arr[0];

      // Figure out how many blocks can be active in each sm.
      hipOccupancyMaxActiveBlocksPerMultiprocessor(
          &num_blocks_per_sm, MomentumLarsKernel<T, MT>, LARS_BLOCK_SIZE,
          sizeof(MT) << 1);
      LarsThreadConfig<float> lars_thread_config(numel, sm_num,
                                                 num_blocks_per_sm);
      int repeat_times = lars_thread_config.GetRepeatTimes(numel);
      int thresh = 0;
      void* cuda_param[] = {
          reinterpret_cast<void*>(&param_data),
          reinterpret_cast<void*>(&grad_data),
          reinterpret_cast<void*>(&velocity_data),
          reinterpret_cast<void*>(&param_out_data),
          reinterpret_cast<void*>(&velocity_out_data),
          reinterpret_cast<void*>(&master_param_data),
          reinterpret_cast<void*>(&master_param_out_data),
          reinterpret_cast<void*>(&lr),
          reinterpret_cast<void*>(&p_buffer),
          reinterpret_cast<void*>(&g_buffer),
          reinterpret_cast<void*>(&mu),
          reinterpret_cast<void*>(&lars_coeff),
          reinterpret_cast<void*>(&lars_weight_decay),
          reinterpret_cast<void*>(&epsilon),
          reinterpret_cast<void*>(&rescale_grad),
          reinterpret_cast<void*>(&repeat_times),
          reinterpret_cast<void*>(&thresh),  // Just a placeholder
          reinterpret_cast<void*>(&numel),
          reinterpret_cast<void*>(&multi_precision)};
      // Lanuch all sm theads.
      hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
          reinterpret_cast<void*>(MomentumLarsKernel<T), MT>),
          lars_thread_config.grid_for_lars, LARS_BLOCK_SIZE, cuda_param, 0,
          cuda_ctx.stream());
    }
#else
    for (int i = 0; i < op_num; ++i) {
      const MT* master_param_data =
          multi_precision ? master_param[i]->data<MT>() : nullptr;
      MT* master_param_out_data =
          multi_precision
              ? master_param_out[i]->mutable_data<MT>(ctx.GetPlace())
              : nullptr;
      SeparatedLarsMomentumOpCUDAKernel<T, MT>(
          cuda_ctx, param[i]->data<T>(),
          param_out[i]->mutable_data<T>(ctx.GetPlace()),
          velocity[i]->data<MT>(),
          velocity_out[i]->mutable_data<MT>(ctx.GetPlace()), grad[i]->data<T>(),
          learning_rate[i]->data<MT>(), p_buffer, g_buffer, mu, lars_coeff,
          weight_decay_arr[i], epsilon, rescale_grad, param[i]->numel(),
          master_param_data, master_param_out_data, multi_precision);
    }
#endif
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    lars_momentum,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::LarsMomentumOpCUDAKernel<paddle::platform::CUDADeviceContext,
                                  paddle::platform::float16>);
