#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/fluid/operators/optimizers/adagrad_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

namespace {

template <typename T, int block_size>
__global__ void MergeGradKernel(const T* grad, const int64_t* grad_rows,
                                T* grad_merge, const int64_t* grad_merge_rows,
                                size_t grad_merge_rows_size,
                                int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;
  __shared__ size_t grad_merge_idx;

  if (tid == 0) {
    for (size_t i = 0; i < grad_merge_rows_size; i++) {
      if (grad_rows[ty] == grad_merge_rows[i]) {
        grad_merge_idx = i;
      }
    }
  }

  __syncthreads();

  grad += ty * row_numel;
  grad_merge += grad_merge_idx * row_numel;
  for (int index = tid; index < row_numel; index += block_size) {
    paddle::platform::CudaAtomicAdd(grad_merge + index, grad[index]);
  }
}

template <typename T, int block_size>
__global__ void SparseAdagradFunctorKernel(const T* grad, const int64_t* rows,
                                           const T* learning_rate, T* param,
                                           T* moment, int64_t row_numel,
                                           T epsilon) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  grad += ty * row_numel;
  param += rows[ty] * row_numel;
  moment += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(param + index,
                                    -1.0 * learning_rate[0] * grad[index] /
                                        (sqrt(moment[index]) + epsilon));
  }
}
}  // namespace

template <typename T>
struct SparseAdagradFunctor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& grad,
                  const framework::Tensor& learning_rate, T epsilon,
                  framework::Tensor* moment, framework::Tensor* param) {
    // 1. g_m.rows = set(g.rows)
    auto grad_width = grad.value().dims()[1];
    math::scatter::MergeAdd<platform::CUDADeviceContext, T> merge_func;
    auto grad_merge = merge_func(context, grad);
    auto* grad_merge_data = grad_merge.mutable_value()->template data<T>();
    framework::Vector<int64_t> merge_rows(grad_merge.rows());
    // 2. m += g_m * g_m
    auto grad_square =
        SquareSelectedRows<platform::CUDADeviceContext, T>(context, grad_merge);

    math::SelectedRowsAddToTensor<platform::CUDADeviceContext, T> functor;
    functor(context, grad_square, moment);

    // 3. update parameter
    auto* lr = learning_rate.data<T>();
    auto* param_data = param->data<T>();
    auto* moment_data = moment->data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid2(1, merge_rows.size());
    SparseAdagradFunctorKernel<
        T, 256><<<grid2, threads, 0,
                  reinterpret_cast<const platform::CUDADeviceContext&>(context)
                      .stream()>>>(
        grad_merge_data, merge_rows.CUDAMutableData(context.GetPlace()), lr,
        param_data, moment_data, grad_width, epsilon);
  }
};

template struct SparseAdagradFunctor<platform::CUDADeviceContext, float>;
template struct SparseAdagradFunctor<platform::CUDADeviceContext, double>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    adagrad, ops::AdagradOpKernel<paddle::platform::CUDADeviceContext, float>,
    ops::AdagradOpKernel<paddle::platform::CUDADeviceContext, double>);
