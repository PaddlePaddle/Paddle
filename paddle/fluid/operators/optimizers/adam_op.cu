#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/operators/optimizers/adam_op.h"
#include "paddle/fluid/platform/float16.h"

namespace paddle {
namespace operators {

template <typename T, typename MT>
__global__ void AdamKernelREG(MT beta1, MT beta2, MT epsilon, MT beta1_pow_,
                              MT beta2_pow_, const MT* moment1, MT* moment1_out,
                              const MT* moment2, MT* moment2_out, const MT* lr_,
                              const T* grad, const T* param, T* param_out,
                              const MT* master_param, MT* master_param_out,
                              int ndim) {
  MT lr = *lr_;
  MT beta1_pow = beta1_pow_;
  MT beta2_pow = beta2_pow_;

  lr *= sqrt(static_cast<MT>(1.0) - beta2_pow) /
        (static_cast<MT>(1.0) - beta1_pow);

  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (; id < ndim; id += gridDim.x * blockDim.x) {
    MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
    MT g = static_cast<MT>(grad[id]);
    MT mom1 = moment1[id];
    MT mom2 = moment2[id];
    mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
    mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g * g;
    p -= lr * (mom1 /
               (sqrt(mom2) + epsilon * sqrt(static_cast<MT>(1.0) - beta2_pow)));

    moment1_out[id] = mom1;
    moment2_out[id] = mom2;
    param_out[id] = static_cast<T>(p);
    if (master_param_out) {
      master_param_out[id] = p;
    }
  }
}

template <typename T, typename MT>
__global__ void AdamKernelMEM(MT beta1, MT beta2, MT epsilon,
                              const MT* beta1_pow_, const MT* beta2_pow_,
                              const MT* moment1, MT* moment1_out,
                              const MT* moment2, MT* moment2_out, const MT* lr_,
                              const T* grad, const T* param, T* param_out,
                              const MT* master_param, MT* master_param_out,
                              int ndim) {
  MT lr = *lr_;
  MT beta1_pow = *beta1_pow_;
  MT beta2_pow = *beta2_pow_;

  lr *= sqrt(static_cast<MT>(1.0) - beta2_pow) /
        (static_cast<MT>(1.0) - beta1_pow);

  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (; id < ndim; id += gridDim.x * blockDim.x) {
    MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
    MT g = static_cast<MT>(grad[id]);
    MT mom1 = static_cast<MT>(moment1[id]);
    MT mom2 = static_cast<MT>(moment2[id]);
    mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
    mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g * g;
    p -= lr * (mom1 /
               (sqrt(mom2) + epsilon * sqrt(static_cast<MT>(1.0) - beta2_pow)));

    moment1_out[id] = mom1;
    moment2_out[id] = mom2;
    param_out[id] = static_cast<T>(p);
    if (master_param_out) {
      master_param_out[id] = p;
    }
  }
}
template <typename T>
__global__ void UpdateBetaPow(T beta1, T beta2, const T* beta1_pow_,
                              const T* beta2_pow_, T* beta1_pow_out,
                              T* beta2_pow_out) {
  *beta1_pow_out = beta1 * beta1_pow_[0];
  *beta2_pow_out = beta2 * beta2_pow_[0];
}

template <typename T, typename MT>
__global__ void SparseAdamCUDAKernelREG(
    MT beta1, MT beta2, MT epsilon, const MT beta1_pow, const MT beta2_pow,
    const MT* mom1_, MT* mom1_out_, const MT* mom2_, MT* mom2_out_,
    const MT* lr_, const T* grad_, const T* param_, T* param_out_,
    const MT* master_param, MT* master_param_out, const int64_t* rows_,
    int64_t row_numel, int64_t row_count, bool lazy_mode, int ndim) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  MT lr = *lr_;
  lr *= sqrt(static_cast<MT>(1.0) - beta2_pow) /
        (static_cast<MT>(1.0) - beta1_pow);

  for (; id < ndim; id += blockDim.x * gridDim.x) {
    auto row_idx =
        math::BinarySearch<int64_t>(rows_, row_count, id / row_numel);
    if (lazy_mode && row_idx < 0) {
      return;
    } else {
      MT mom1 = mom1_[id];
      MT mom2 = mom2_[id];
      MT p = master_param ? master_param[id] : static_cast<MT>(param_[id]);
      MT g = row_idx >= 0
                 ? static_cast<MT>(grad_[row_idx * row_numel + id % row_numel])
                 : static_cast<MT>(0);
      mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
      mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g * g;
      p -= lr * (mom1 / (sqrt(mom2) +
                         epsilon * sqrt(static_cast<MT>(1.0) - beta2_pow)));

      // Write back to global memory
      mom1_out_[id] = mom1;
      mom2_out_[id] = mom2;
      param_out_[id] = static_cast<T>(p);
      if (master_param_out) {
        master_param_out[id] = p;
      }
    }
  }
}

template <typename T>
class AdamOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const auto* param_var = ctx.InputVar("Param");
    PADDLE_ENFORCE_EQ(param_var->IsType<framework::LoDTensor>(), true,
                      platform::errors::InvalidArgument(
                          "The Var(%s)'s type should be LoDTensor, "
                          "but the received is %s",
                          ctx.InputNames("Param").front(),
                          framework::ToTypeName(param_var->Type())));

    using paddle::framework::LoDTensor;
    using MPDType = typename details::MPTypeTrait<T>::Type;

    int64_t min_row_size_to_use_multithread =
        ctx.Attr<int64_t>("min_row_size_to_use_multithread");
    bool lazy_mode = ctx.Attr<bool>("lazy_mode");
    MPDType epsilon = static_cast<MPDType>(ctx.Attr<float>("epsilon"));
    auto* param = ctx.Input<LoDTensor>("Param");
    auto* grad_var = ctx.InputVar("Grad");
    auto* mom1 = ctx.Input<LoDTensor>("Moment1");
    auto* mom2 = ctx.Input<LoDTensor>("Moment2");
    auto* lr = ctx.Input<LoDTensor>("LearningRate");

    auto* beta1_pow = ctx.Input<LoDTensor>("Beta1Pow");
    auto* beta2_pow = ctx.Input<LoDTensor>("Beta2Pow");

    auto* param_out = ctx.Output<LoDTensor>("ParamOut");
    auto* mom1_out = ctx.Output<LoDTensor>("Moment1Out");
    auto* mom2_out = ctx.Output<LoDTensor>("Moment2Out");
    auto* beta1_pow_out = ctx.Output<LoDTensor>("Beta1PowOut");
    auto* beta2_pow_out = ctx.Output<LoDTensor>("Beta2PowOut");

    MPDType beta1 = static_cast<MPDType>(ctx.Attr<float>("beta1"));
    if (ctx.HasInput("Beta1Tensor")) {
      auto* beta1_tensor = ctx.Input<framework::Tensor>("Beta1Tensor");
      PADDLE_ENFORCE_EQ(beta1_tensor->numel(), 1,
                        platform::errors::InvalidArgument(
                            "Input(Beta1Tensor) size must be 1, but get %d",
                            beta1_tensor->numel()));
      beta1 = static_cast<MPDType>(GetAttrFromTensor(beta1_tensor));
    }
    MPDType beta2 = static_cast<MPDType>(ctx.Attr<float>("beta2"));
    if (ctx.HasInput("Beta2Tensor")) {
      auto* beta2_tensor = ctx.Input<framework::Tensor>("Beta2Tensor");
      PADDLE_ENFORCE_EQ(beta2_tensor->numel(), 1,
                        platform::errors::InvalidArgument(
                            "Input(Beta2Tensor) size must be 1, but get %d",
                            beta2_tensor->numel()));
      beta2 = static_cast<MPDType>(GetAttrFromTensor(beta2_tensor));
    }
    VLOG(3) << "beta1_pow.numel() : " << beta1_pow->numel()
            << "beta2_pow.numel() : " << beta2_pow->numel();
    VLOG(3) << "param.numel(): " << param->numel();
    PADDLE_ENFORCE_EQ(beta1_pow_out->numel(), 1,
                      platform::errors::InvalidArgument(
                          "beta1 pow output size should be 1, but received "
                          "value is:%d.",
                          beta1_pow_out->numel()));

    PADDLE_ENFORCE_EQ(beta2_pow_out->numel(), 1,
                      platform::errors::InvalidArgument(
                          "beta2 pow output size should be 1, but received "
                          "value is:%d.",
                          beta2_pow_out->numel()));

    const bool multi_precision = ctx.Attr<bool>("multi_precision");
    const LoDTensor* master_param = nullptr;
    LoDTensor* master_param_out = nullptr;
    if (multi_precision) {
      bool has_master =
          ctx.HasInput("MasterParam") && ctx.HasOutput("MasterParamOut");
      PADDLE_ENFORCE_EQ(has_master, true,
                        platform::errors::InvalidArgument(
                            "The Input(MasterParam) and Output(MasterParamOut) "
                            "should not be null when "
                            "the attr `multi_precision` is true"));
      master_param = ctx.Input<LoDTensor>("MasterParam");
      master_param_out = ctx.Output<LoDTensor>("MasterParamOut");
    }
    const MPDType* master_in_data =
        multi_precision ? master_param->data<MPDType>() : nullptr;
    MPDType* master_out_data =
        multi_precision
            ? master_param_out->mutable_data<MPDType>(ctx.GetPlace())
            : nullptr;

    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    if (grad_var->IsType<framework::LoDTensor>()) {
      auto* grad = ctx.Input<LoDTensor>("Grad");

      // update param and moment
      int threads = 512;
      int blocks = (param->numel() + threads - 1) / threads;

      if (beta1_pow->place() == platform::CPUPlace() &&
          beta2_pow->place() == platform::CPUPlace()) {
        // Compute with betapow in REG
        AdamKernelREG<T, MPDType><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1, beta2, epsilon, *beta1_pow->data<MPDType>(),
            *beta2_pow->data<MPDType>(), mom1->data<MPDType>(),
            mom1_out->mutable_data<MPDType>(ctx.GetPlace()),
            mom2->data<MPDType>(),
            mom2_out->mutable_data<MPDType>(ctx.GetPlace()),
            lr->data<MPDType>(), grad->data<T>(), param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), master_in_data,
            master_out_data, param->numel());
        // Cpu update
        beta1_pow_out->mutable_data<MPDType>(platform::CPUPlace())[0] =
            beta1 * beta1_pow->data<MPDType>()[0];
        beta2_pow_out->mutable_data<MPDType>(platform::CPUPlace())[0] =
            beta2 * beta2_pow->data<MPDType>()[0];
      } else {
        AdamKernelMEM<T, MPDType><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1, beta2, epsilon, beta1_pow->data<MPDType>(),
            beta2_pow->data<MPDType>(), mom1->data<MPDType>(),
            mom1_out->mutable_data<MPDType>(ctx.GetPlace()),
            mom2->data<MPDType>(),
            mom2_out->mutable_data<MPDType>(ctx.GetPlace()),
            lr->data<MPDType>(), grad->data<T>(), param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), master_in_data,
            master_out_data, param->numel());
        // Update with gpu
        UpdateBetaPow<MPDType><<<1, 32, 0, dev_ctx.stream()>>>(
            beta1, beta2, beta1_pow->data<MPDType>(),
            beta2_pow->data<MPDType>(),
            beta1_pow_out->mutable_data<MPDType>(ctx.GetPlace()),
            beta2_pow_out->mutable_data<MPDType>(ctx.GetPlace()));
      }

    } else if (grad_var->IsType<framework::SelectedRows>()) {
      auto* grad = ctx.Input<framework::SelectedRows>("Grad");
      if (grad->rows().size() == 0) {
        VLOG(3) << "grad row size is 0!!";
        return;
      }

      std::vector<int64_t> cpu_rows(grad->rows().begin(), grad->rows().end());
      bool is_strict_sorted = true;
      for (size_t i = 1; i < cpu_rows.size(); ++i) {
        if (cpu_rows[i - 1] >= cpu_rows[i]) {
          is_strict_sorted = false;
          break;
        }
      }

      framework::SelectedRows tmp_grad_merge;
      const framework::SelectedRows* grad_merge_ptr;
      if (is_strict_sorted) {
        grad_merge_ptr = grad;
      } else {
        // merge duplicated rows if any.
        // The rows of grad_merge have been sorted inside MergeAdd functor
        scatter::MergeAdd<platform::CUDADeviceContext, T> merge_func;
        merge_func(ctx.template device_context<platform::CUDADeviceContext>(),
                   *grad, &tmp_grad_merge, true);
        grad_merge_ptr = &tmp_grad_merge;
      }
      auto& grad_merge = *grad_merge_ptr;
      auto& grad_tensor = grad_merge.value();
      const T* grad_data = grad_tensor.template data<T>();
      const int64_t* rows = grad_merge.rows().Data(ctx.GetPlace());
      auto row_numel = grad_tensor.numel() / grad_merge.rows().size();

      if (beta1_pow->place() == platform::CPUPlace() &&
          beta2_pow->place() == platform::CPUPlace()) {
        int threads = 512;
        int ndim = param->numel();
        int blocks = (ndim + threads - 1) / threads;

        SparseAdamCUDAKernelREG<
            T, MPDType><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1, beta2, epsilon, *beta1_pow->data<MPDType>(),
            *beta2_pow->data<MPDType>(), mom1->data<MPDType>(),
            mom1_out->mutable_data<MPDType>(ctx.GetPlace()),
            mom2->data<MPDType>(),
            mom2_out->mutable_data<MPDType>(ctx.GetPlace()),
            lr->data<MPDType>(), grad_data, param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), master_in_data,
            master_out_data, rows, row_numel, grad_merge.rows().size(),
            lazy_mode, ndim);
        // Update with cpu
        beta1_pow_out->mutable_data<MPDType>(platform::CPUPlace())[0] =
            beta1 * beta1_pow->data<MPDType>()[0];
        beta2_pow_out->mutable_data<MPDType>(platform::CPUPlace())[0] =
            beta2 * beta2_pow->data<MPDType>()[0];
      } else {
        SparseAdamFunctor<T, GPUAdam, MPDType> functor(
            beta1, beta2, epsilon, beta1_pow->data<MPDType>(),
            beta2_pow->data<MPDType>(), mom1->data<MPDType>(),
            mom1_out->mutable_data<MPDType>(ctx.GetPlace()),
            mom2->data<MPDType>(),
            mom2_out->mutable_data<MPDType>(ctx.GetPlace()),
            lr->data<MPDType>(), grad_data, param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), master_in_data,
            master_out_data, rows, row_numel, grad_merge.rows().size(),
            lazy_mode);

        // FIXME(minqiyang): remove BinarySearch in GPU later
        platform::ForRange<platform::CUDADeviceContext> for_range(
            static_cast<const platform::CUDADeviceContext&>(
                ctx.device_context()),
            param->numel());
        for_range(functor);
        // update beta1 and beta2
        UpdateBetaPow<MPDType><<<1, 32, 0, dev_ctx.stream()>>>(
            beta1, beta2, beta1_pow->data<MPDType>(),
            beta2_pow->data<MPDType>(),
            beta1_pow_out->mutable_data<MPDType>(ctx.GetPlace()),
            beta2_pow_out->mutable_data<MPDType>(ctx.GetPlace()));
      }
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Variable type not supported by adam_op"));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(adam, ops::AdamOpCUDAKernel<float>,
                        ops::AdamOpCUDAKernel<double>,
                        ops::AdamOpCUDAKernel<plat::float16>);
