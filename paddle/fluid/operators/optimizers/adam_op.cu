#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/operators/optimizers/adam_op.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void AdamKernelREG(T beta1, T beta2, T epsilon, T beta1_pow_,
                              T beta2_pow_, const T* moment1, T* moment1_out,
                              const T* moment2, T* moment2_out, const T* lr_,
                              const T* grad, const T* param, T* param_out,
                              int ndim) {
  T lr = *lr_;
  T beta1_pow = beta1_pow_;
  T beta2_pow = beta2_pow_;

  lr *=
      sqrt(static_cast<T>(1.0) - beta2_pow) / (static_cast<T>(1.0) - beta1_pow);

  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (; id < ndim; id += gridDim.x * blockDim.x) {
    T p = param[id];
    T g = grad[id];
    T mom1 = moment1[id];
    T mom2 = moment2[id];
    mom1 = beta1 * mom1 + (static_cast<T>(1.0) - beta1) * g;
    mom2 = beta2 * mom2 + (static_cast<T>(1.0) - beta2) * g * g;
    p -= lr * (mom1 / (sqrt(mom2) + epsilon));

    moment1_out[id] = mom1;
    moment2_out[id] = mom2;
    param_out[id] = p;
  }
}

template <typename T>
__global__ void AdamKernelMEM(T beta1, T beta2, T epsilon, const T* beta1_pow_,
                              const T* beta2_pow_, const T* moment1,
                              T* moment1_out, const T* moment2, T* moment2_out,
                              const T* lr_, const T* grad, const T* param,
                              T* param_out, int ndim) {
  T lr = *lr_;
  T beta1_pow = *beta1_pow_;
  T beta2_pow = *beta2_pow_;

  lr *=
      sqrt(static_cast<T>(1.0) - beta2_pow) / (static_cast<T>(1.0) - beta1_pow);

  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (; id < ndim; id += gridDim.x * blockDim.x) {
    T p = param[id];
    T g = grad[id];
    T mom1 = moment1[id];
    T mom2 = moment2[id];
    mom1 = beta1 * mom1 + (static_cast<T>(1.0) - beta1) * g;
    mom2 = beta2 * mom2 + (static_cast<T>(1.0) - beta2) * g * g;
    p -= lr * (mom1 / (sqrt(mom2) + epsilon));

    moment1_out[id] = mom1;
    moment2_out[id] = mom2;
    param_out[id] = p;
  }
}
template <typename T>
__global__ void UpdateBetaPow(T beta1, T beta2, const T* beta1_pow_,
                              const T* beta2_pow_, T* beta1_pow_out,
                              T* beta2_pow_out) {
  *beta1_pow_out = beta1 * beta1_pow_[0];
  *beta2_pow_out = beta2 * beta2_pow_[0];
}

template <typename T>
__global__ void SparseAdamCUDAKernelREG(
    T beta1, T beta2, T epsilon, const T beta1_pow, const T beta2_pow,
    const T* mom1_, T* mom1_out_, const T* mom2_, T* mom2_out_, const T* lr_,
    const T* grad_, const T* param_, T* param_out_, const int64_t* rows_,
    int64_t row_numel, int64_t row_count, bool lazy_mode, int ndim) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  T lr = *lr_;
  lr *= sqrt(1 - beta2_pow) / (1 - beta1_pow);

  for (; id < ndim; id += blockDim.x * gridDim.x) {
    auto row_idx =
        math::BinarySearch<int64_t>(rows_, row_count, id / row_numel);
    if (lazy_mode && row_idx < 0) {
      return;
    } else {
      T mom1 = mom1_[id];
      T mom2 = mom2_[id];
      T p = param_[id];
      T g = row_idx >= 0 ? grad_[row_idx * row_numel + id % row_numel] : 0;
      mom1 = beta1 * mom1 + (1 - beta1) * g;
      mom2 = beta2 * mom2 + (1 - beta2) * g * g;
      p -= lr * (mom1 / (sqrt(mom2) + epsilon));

      // Write back to global memory
      mom1_out_[id] = mom1;
      mom2_out_[id] = mom2;
      param_out_[id] = p;
    }
  }
}

template <typename T>
class AdamOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    const auto* param_var = ctx.InputVar("Param");
    PADDLE_ENFORCE_EQ(param_var->IsType<framework::LoDTensor>(), true,
                      platform::errors::InvalidArgument(
                          "The Var(%s)'s type should be LoDTensor, "
                          "but the received is %s",
                          ctx.InputNames("Param").front(),
                          framework::ToTypeName(param_var->Type())));

    using paddle::framework::LoDTensor;

    int64_t min_row_size_to_use_multithread =
        ctx.Attr<int64_t>("min_row_size_to_use_multithread");
    bool lazy_mode = ctx.Attr<bool>("lazy_mode");
    T epsilon = static_cast<T>(ctx.Attr<float>("epsilon"));
    auto* param = ctx.Input<LoDTensor>("Param");
    auto* grad_var = ctx.InputVar("Grad");
    auto* mom1 = ctx.Input<LoDTensor>("Moment1");
    auto* mom2 = ctx.Input<LoDTensor>("Moment2");
    auto* lr = ctx.Input<LoDTensor>("LearningRate");

    auto* beta1_pow = ctx.Input<LoDTensor>("Beta1Pow");
    auto* beta2_pow = ctx.Input<LoDTensor>("Beta2Pow");

    auto* param_out = ctx.Output<LoDTensor>("ParamOut");
    auto* mom1_out = ctx.Output<LoDTensor>("Moment1Out");
    auto* mom2_out = ctx.Output<LoDTensor>("Moment2Out");
    auto* beta1_pow_out = ctx.Output<LoDTensor>("Beta1PowOut");
    auto* beta2_pow_out = ctx.Output<LoDTensor>("Beta2PowOut");

    T beta1 = static_cast<T>(ctx.Attr<float>("beta1"));
    if (ctx.HasInput("Beta1Tensor")) {
      auto* beta1_tensor = ctx.Input<framework::Tensor>("Beta1Tensor");
      PADDLE_ENFORCE_EQ(beta1_tensor->numel(), 1,
                        platform::errors::InvalidArgument(
                            "Input(Beta1Tensor) size must be 1, but get %d",
                            beta1_tensor->numel()));
      beta1 = static_cast<T>(GetAttrFromTensor(beta1_tensor));
    }
    T beta2 = static_cast<T>(ctx.Attr<float>("beta2"));
    if (ctx.HasInput("Beta2Tensor")) {
      auto* beta2_tensor = ctx.Input<framework::Tensor>("Beta2Tensor");
      PADDLE_ENFORCE_EQ(beta2_tensor->numel(), 1,
                        platform::errors::InvalidArgument(
                            "Input(Beta2Tensor) size must be 1, but get %d",
                            beta2_tensor->numel()));
      beta2 = static_cast<T>(GetAttrFromTensor(beta2_tensor));
    }
    VLOG(3) << "beta1_pow.numel() : " << beta1_pow->numel()
            << "beta2_pow.numel() : " << beta2_pow->numel();
    VLOG(3) << "param.numel(): " << param->numel();
    PADDLE_ENFORCE_EQ(beta1_pow_out->numel(), 1,
                      platform::errors::InvalidArgument(
                          "beta1 pow output size should be 1, but received "
                          "value is:%d.",
                          beta1_pow_out->numel()));

    PADDLE_ENFORCE_EQ(beta2_pow_out->numel(), 1,
                      platform::errors::InvalidArgument(
                          "beta2 pow output size should be 1, but received "
                          "value is:%d.",
                          beta2_pow_out->numel()));
    auto& dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();

    if (grad_var->IsType<framework::LoDTensor>()) {
      auto* grad = ctx.Input<LoDTensor>("Grad");

      // update param and moment
      int threads = 512;
      int blocks = (param->numel() + threads - 1) / threads;

      if (beta1_pow->place() == platform::CPUPlace() &&
          beta2_pow->place() == platform::CPUPlace()) {
        // Compute with betapow in REG
        AdamKernelREG<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1, beta2, epsilon, *beta1_pow->data<T>(), *beta2_pow->data<T>(),
            mom1->data<T>(), mom1_out->mutable_data<T>(ctx.GetPlace()),
            mom2->data<T>(), mom2_out->mutable_data<T>(ctx.GetPlace()),
            lr->data<T>(), grad->data<T>(), param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), param->numel());
        // Cpu update
        beta1_pow_out->mutable_data<T>(platform::CPUPlace())[0] =
            beta1 * beta1_pow->data<T>()[0];
        beta2_pow_out->mutable_data<T>(platform::CPUPlace())[0] =
            beta2 * beta2_pow->data<T>()[0];
      } else {
        AdamKernelMEM<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1, beta2, epsilon, beta1_pow->data<T>(), beta2_pow->data<T>(),
            mom1->data<T>(), mom1_out->mutable_data<T>(ctx.GetPlace()),
            mom2->data<T>(), mom2_out->mutable_data<T>(ctx.GetPlace()),
            lr->data<T>(), grad->data<T>(), param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), param->numel());
        // Update with gpu
        UpdateBetaPow<T><<<1, 32, 0, dev_ctx.stream()>>>(
            beta1, beta2, beta1_pow->data<T>(), beta2_pow->data<T>(),
            beta1_pow_out->mutable_data<T>(ctx.GetPlace()),
            beta2_pow_out->mutable_data<T>(ctx.GetPlace()));
      }

    } else if (grad_var->IsType<framework::SelectedRows>()) {
      auto* grad = ctx.Input<framework::SelectedRows>("Grad");
      if (grad->rows().size() == 0) {
        VLOG(3) << "grad row size is 0!!";
        return;
      }

      std::vector<int64_t> cpu_rows(grad->rows().begin(), grad->rows().end());
      bool is_strict_sorted = true;
      for (size_t i = 1; i < cpu_rows.size(); ++i) {
        if (cpu_rows[i - 1] >= cpu_rows[i]) {
          is_strict_sorted = false;
          break;
        }
      }

      framework::SelectedRows tmp_grad_merge;
      const framework::SelectedRows* grad_merge_ptr;
      if (is_strict_sorted) {
        grad_merge_ptr = grad;
      } else {
        // merge duplicated rows if any.
        // The rows of grad_merge have been sorted inside MergeAdd functor
        scatter::MergeAdd<platform::CUDADeviceContext, T> merge_func;
        merge_func(ctx.template device_context<platform::CUDADeviceContext>(),
                   *grad, &tmp_grad_merge, true);
        grad_merge_ptr = &tmp_grad_merge;
      }
      auto& grad_merge = *grad_merge_ptr;
      auto& grad_tensor = grad_merge.value();
      const T* grad_data = grad_tensor.template data<T>();
      const int64_t* rows = grad_merge.rows().Data(ctx.GetPlace());
      auto row_numel = grad_tensor.numel() / grad_merge.rows().size();

      if (beta1_pow->place() == platform::CPUPlace() &&
          beta2_pow->place() == platform::CPUPlace()) {
        int threads = 512;
        int ndim = param->numel();
        int blocks = (ndim + threads - 1) / threads;

        SparseAdamCUDAKernelREG<T><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1, beta2, epsilon, *beta1_pow->data<T>(), *beta2_pow->data<T>(),
            mom1->data<T>(), mom1_out->mutable_data<T>(ctx.GetPlace()),
            mom2->data<T>(), mom2_out->mutable_data<T>(ctx.GetPlace()),
            lr->data<T>(), grad_data, param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), rows, row_numel,
            grad_merge.rows().size(), lazy_mode, ndim);
        // Update with cpu
        beta1_pow_out->mutable_data<T>(platform::CPUPlace())[0] =
            beta1 * beta1_pow->data<T>()[0];
        beta2_pow_out->mutable_data<T>(platform::CPUPlace())[0] =
            beta2 * beta2_pow->data<T>()[0];
      } else {
        SparseAdamFunctor<T, GPUAdam> functor(
            beta1, beta2, epsilon, beta1_pow->data<T>(), beta2_pow->data<T>(),
            mom1->data<T>(), mom1_out->mutable_data<T>(ctx.GetPlace()),
            mom2->data<T>(), mom2_out->mutable_data<T>(ctx.GetPlace()),
            lr->data<T>(), grad_data, param->data<T>(),
            param_out->mutable_data<T>(ctx.GetPlace()), rows, row_numel,
            grad_merge.rows().size(), lazy_mode);

        // FIXME(minqiyang): remove BinarySearch in GPU later
        platform::ForRange<platform::CUDADeviceContext> for_range(
            static_cast<const platform::CUDADeviceContext&>(
                ctx.device_context()),
            param->numel());
        for_range(functor);
        // update beta1 and beta2
        UpdateBetaPow<T><<<1, 32, 0, dev_ctx.stream()>>>(
            beta1, beta2, beta1_pow->data<T>(), beta2_pow->data<T>(),
            beta1_pow_out->mutable_data<T>(ctx.GetPlace()),
            beta2_pow_out->mutable_data<T>(ctx.GetPlace()));
      }
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Variable type not supported by adam_op"));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(adam, ops::AdamOpCUDAKernel<float>,
                        ops::AdamOpCUDAKernel<double>);
