#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/details/var_utils.h"
#include "paddle/fluid/framework/details/var_utils_detail.h"

#include <algorithm>
#include <unordered_map>
#include <utility>

namespace paddle {
namespace framework {
namespace details {

static std::unordered_map<std::string, memory::AllocationPtr> op_var2gpu_str;
static std::mutex op_var2gpu_str_mutex;

template <typename T>
__device__ __forceinline__ void PrintNanInfKernel(const T* value,
                                                  const size_t numel,
                                                  int print_num,
                                                  char* debug_info) {
  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ unsigned int nan_count, inf_count, num_count;
  if (threadIdx.x == 0) nan_count = inf_count = num_count = 0;
  __syncthreads;

  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    unsigned int count = 0;
    if (isnan(value[i])) {
      count = atomicAdd(&nan_count, 1);
    } else if (isinf(value[i])) {
      count = atomicAdd(&inf_count, 1);
    } else {
      count = atomicAdd(&num_count, 1);
    }
    // for cuda, print in every block
    if (count < print_num) {
      printf("numel:%lu idx:%lu value:%f\n", static_cast<uint64_t>(numel),
             static_cast<uint64_t>(i), static_cast<float>(value[i]));
    }
  }
  __syncthreads;

  if (true && threadIdx.x == 0) {
    printf("In block %d, there has %u,%u,%u nan,inf,num\n", blockIdx.x,
           nan_count, inf_count, num_count);
    PADDLE_ENFORCE(false, "===ERROR: in %s find nan or inf===", debug_info);
  }
}

// Resnet 2gpus speed test, no check 270 images/s, this check 229 images/s
template <typename T>
__global__ void CheckNanInfKernel(const T* value, const size_t numel,
                                  int print_num, char* debug_info) {
  /// step 1, judge wheater has nan or inf
  __shared__ volatile int has_nan_inf;
  if (threadIdx.x == 0) has_nan_inf = false;
  __syncthreads();

  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  T sum = static_cast<T>(0.0);
  // Todo(wangxi). simd speed up
  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    sum += (value[i] - value[i]);
  }

  if (isnan(sum) || isinf(sum)) has_nan_inf = true;
  __syncthreads();

  /// Note. different blocks may behave differently
  if (!has_nan_inf) return;

  PrintNanInfKernel(value, numel, print_num, debug_info);
}

template <>
template <typename T>
void TensorCheckerVisitor<platform::CUDADeviceContext>::apply(
    typename std::enable_if<std::is_floating_point<T>::value>::type*) const {
  int print_num = 3;

  auto* dev_ctx = reinterpret_cast<platform::CUDADeviceContext*>(
      platform::DeviceContextPool::Instance().Get(tensor_.place()));

  std::string op_var = "[op=" + op_type_ + "] [tensor=" + var_name_ + "]";
  char* gpu_str_ptr = NULL;

  {
    std::lock_guard<std::mutex> guard(op_var2gpu_str_mutex);
    if (op_var2gpu_str.find(op_var) == op_var2gpu_str.end()) {
      auto gpu_str_tensor =
          paddle::memory::Alloc(*dev_ctx, op_var.length() + 1);
      gpu_str_ptr = reinterpret_cast<char*>(gpu_str_tensor->ptr());

      op_var2gpu_str.emplace(op_var, std::move(gpu_str_tensor));

      auto iter = op_var2gpu_str.find(op_var);
      PADDLE_ENFORCE_EQ(iter != op_var2gpu_str.end(), true);

      PADDLE_ENFORCE_CUDA_SUCCESS(
          hipMemcpyAsync(gpu_str_ptr, iter->first.c_str(), op_var.length() + 1,
                          hipMemcpyHostToDevice, dev_ctx->stream()));
    } else {
      auto iter = op_var2gpu_str.find(op_var);
      PADDLE_ENFORCE_EQ(iter != op_var2gpu_str.end(), true);
      gpu_str_ptr = reinterpret_cast<char*>(iter->second->ptr());
    }
  }

  const size_t threads = 1024;
  size_t blocks = std::min(128ul, (tensor_.numel() + threads - 1) / threads);
  CheckNanInfKernel<<<blocks, threads, 0, dev_ctx->stream()>>>(
      tensor_.data<T>(), tensor_.numel(), print_num, gpu_str_ptr);
}

template <>
void tensor_check<platform::CUDADeviceContext>(const std::string& op_type,
                                               const std::string& var_name,
                                               const framework::Tensor& tensor,
                                               const platform::Place& place) {
  TensorCheckerVisitor<platform::CUDADeviceContext> vistor(op_type, var_name,
                                                           tensor, place);
  VisitDataType(tensor.type(), vistor);
}

}  // namespace details
}  // namespace framework
}  // namespace paddle
