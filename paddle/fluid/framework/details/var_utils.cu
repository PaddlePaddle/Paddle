#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/details/var_utils.h"

#include <algorithm>

#include "paddle/fluid/framework/selected_rows.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#include "paddle/fluid/platform/for_range.h"

namespace paddle {
namespace framework {
namespace details {

// print the first 16 letters of the op name
#define MAX_LEN_OP_NAME 16

// print the first 48 letters of the tensor name
#define MAX_LEN_TENSOR_NAME 48

// Resnet Speed. No check 270, check without DebugInfo 229, check with DebugInfo
// 190.
// Maybe can use id or hash value to reduce DebugInfo size.
struct DebugInfo {
  char op_name[MAX_LEN_OP_NAME];
  char tensor_name[MAX_LEN_TENSOR_NAME];
};

static_assert(sizeof(DebugInfo) == (MAX_LEN_OP_NAME + MAX_LEN_TENSOR_NAME),
              "sizeof(DebugInfo) not aligned");

template <typename T>
__global__ void CheckNanInfKernel(const T* value, const size_t numel,
                                  int print_value, struct DebugInfo info) {
  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  T sum = static_cast<T>(0.0);
  // Todo(wangxi). simd speed up
  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    sum += (value[i] - value[i]);
  }

  if (isnan(sum) || isinf(sum)) {
  } else {
    return;
  }

  if (print_value) {
    for (size_t i = tid; i < numel; i += blockDim.x) {
      if (isnan(value[i]) || isinf(value[i])) {
        printf("idx:%u value:%f\n", i, value[i]);
        // use param control whether print more value
        if (i < numel) {
          printf("idx:%u value:%f\n", i + 1, value[i + 1]);
        }
      }
    }
  }
  __syncthreads();
  // abort or not
  if (true) {
    PADDLE_ENFORCE(0, "===ERROR: in [op=%s] [tensor=%s] find nan or inf===",
                   info.op_name, info.tensor_name);
  }
}

template <typename T>
void CheckNanInf(const T* value, const size_t numel, int print_value,
                 const std::string& op_type, const std::string& var_name) {
  T sum = static_cast<T>(0.0);
#pragma omp parallel for reduction(+ : sum)
  for (int i = 0; i < numel; ++i) {
    sum += (value[i] - value[i]);
  }

  if (std::isnan(sum) || std::isinf(sum)) {
    printf("===ERROR: has nan or inf===");
    if (print_value) {
      for (int i = 0; i < numel; ++i) {
        printf("idx:%u value:%f\n", i, value[i]);
      }
    }
    PADDLE_ENFORCE_EQ(1, 0,
                      "===ERROR: in [op=%s] [tensor=%s] find nan or inf===",
                      op_type, var_name);
  }
}

template <typename DeviceContext>
struct CheckNanInfTool {
  template <typename T>
  void run(const std::string& op_type, const std::string& var_name,
           const framework::Tensor& tensor, const platform::Place& place,
           int print_value,
           typename std::enable_if<std::is_integral<T>::value>::type* = 0);

  template <typename T>
  void run(
      const std::string& op_type, const std::string& var_name,
      const framework::Tensor& tensor, const platform::Place& place,
      int print_value,
      typename std::enable_if<std::is_floating_point<T>::value>::type* = 0);
};

template <typename DeviceContext>
template <typename T>
void CheckNanInfTool<DeviceContext>::run(
    const std::string& op_type, const std::string& var_name,
    const framework::Tensor& tensor, const platform::Place& place,
    int print_value,
    typename std::enable_if<std::is_integral<T>::value>::type*) {
  VLOG(10) << var_name << " need not to check, it's type is not float point";
}

template <>
template <typename T>
void CheckNanInfTool<platform::CUDADeviceContext>::run(
    const std::string& op_type, const std::string& var_name,
    const framework::Tensor& tensor, const platform::Place& place,
    int print_value,
    typename std::enable_if<std::is_floating_point<T>::value>::type*) {
  auto* dev_ctx = reinterpret_cast<platform::CUDADeviceContext*>(
      platform::DeviceContextPool::Instance().Get(tensor.place()));

  DebugInfo debug_info;
  int len_op =
      std::min(MAX_LEN_OP_NAME - 1, static_cast<int>(op_type.length()));
  std::strncpy(debug_info.op_name, op_type.c_str(), len_op);
  debug_info.op_name[len_op] = '\0';

  int len_tensor =
      std::min(MAX_LEN_TENSOR_NAME - 1, static_cast<int>(var_name.length()));
  std::strncpy(debug_info.tensor_name, var_name.c_str(), len_tensor);
  debug_info.tensor_name[len_tensor] = '\0';

  const size_t threads = 1024;
  size_t blocks = std::min(128ul, (tensor.numel() + threads - 1) / threads);
  CheckNanInfKernel<<<blocks, threads, 0, dev_ctx->stream()>>>(
      tensor.data<T>(), tensor.numel(), 1, debug_info);
}

template <>
template <typename T>
void CheckNanInfTool<platform::CPUDeviceContext>::run(
    const std::string& op_type, const std::string& var_name,
    const framework::Tensor& tensor, const platform::Place& place,
    int print_value,
    typename std::enable_if<std::is_floating_point<T>::value>::type*) {
  platform::DeviceContextPool::Instance().Get(tensor.place());

  CheckNanInf(tensor.data<T>(), tensor.numel(), 1, op_type, var_name);
}

struct TensorCheckerVisitor {
  TensorCheckerVisitor(const std::string& op_type, const std::string& var_name,
                       const framework::Tensor& tensor,
                       const platform::Place& place)
      : op_type_(op_type),
        var_name_(var_name),
        tensor_(tensor),
        place_(place) {}

  template <typename T>
  void apply() const {
    int is_print_value = 1;

    if (platform::is_gpu_place(tensor_.place())) {
#ifdef PADDLE_WITH_CUDA
      CheckNanInfTool<platform::CUDADeviceContext> tools;
      tools.run<T>(op_type_, var_name_, tensor_, place_, is_print_value);
#else
      PADDLE_THROW("PaddlePaddle should compile with GPU.");
#endif
      return;
    }

    CheckNanInfTool<platform::CPUDeviceContext> tools;
    tools.run<T>(op_type_, var_name_, tensor_, place_, is_print_value);
  }

  std::string op_type_;
  std::string var_name_;
  const framework::Tensor& tensor_;
  const platform::Place& place_;
};

void EnforceNoNanOrInf(const std::string& op_type,
                       const framework::Scope& scope,
                       const std::string& var_name,
                       const platform::Place& place) {
  auto* var = scope.FindVar(var_name);
  PADDLE_ENFORCE_NOT_NULL(var, "can't find var:%s", var_name);

  const Tensor* tensor{nullptr};
  if (var->IsType<framework::LoDTensor>()) {
    tensor = &var->Get<framework::LoDTensor>();
  } else if (var->IsType<framework::SelectedRows>()) {
    tensor = &var->Get<framework::SelectedRows>().value();
  } else {
    VLOG(10) << var_name << " var_name need not to check";
    return;
  }

  if (tensor->memory_size() == 0) {
    VLOG(10) << var_name << " var_name need not to check, size == 0";
    return;
  }

  VLOG(10) << "begin check " << op_type << " var_name:" << var_name
           << ", place:" << tensor->place() << ", numel:" << tensor->numel();

  TensorCheckerVisitor vistor(op_type, var_name, *tensor, place);
  VisitDataType(tensor->type(), vistor);
}

}  // namespace details
}  // namespace framework
}  // namespace paddle
