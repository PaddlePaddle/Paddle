#include "hip/hip_runtime.h"
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/details/nan_inf_utils.h"
#include "paddle/fluid/framework/details/nan_inf_utils_detail.h"

#include <algorithm>
#include <unordered_map>
#include <utility>
#include <vector>

namespace paddle {
namespace framework {
namespace details {

static std::once_flag init_multi_gpu_op_var_map_flag;

// lazy init
static std::vector<std::unordered_map<std::string, memory::AllocationPtr>>&
multi_op_var2gpu_str() {
  static std::vector<std::unordered_map<std::string, memory::AllocationPtr>>
      _multi_op_var2gpu_str;
  return _multi_op_var2gpu_str;
}

static std::vector<std::mutex>& multi_op_var2gpu_str_mutex() {
  static std::vector<std::mutex> _multi_op_var2gpu_str_mutex;
  return _multi_op_var2gpu_str_mutex;
}

static void InitMultiGPUOpVarMap() {
  int dev_count = platform::GetCUDADeviceCount();
  PADDLE_ENFORCE_GT(dev_count, 0,
                    platform::errors::NotFound(
                        "cuda device must > 0, now dev_count=%d", dev_count));

  // https://stackoverflow.com/questions/16465633/how-can-i-use-something-like-stdvectorstdmutex
  std::vector<std::unordered_map<std::string, memory::AllocationPtr>> tmp_multi(
      dev_count);
  std::vector<std::mutex> tmp_multi_mutex(dev_count);

  multi_op_var2gpu_str().swap(tmp_multi);
  multi_op_var2gpu_str_mutex().swap(tmp_multi_mutex);
}

template <typename T>
__device__ __forceinline__ void PrintNanInfKernel(const T* value,
                                                  const size_t numel,
                                                  int print_num,
                                                  char* debug_info) {
  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ unsigned int nan_count, inf_count, num_count;
  if (threadIdx.x == 0) nan_count = inf_count = num_count = 0;
  __syncthreads;

  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    unsigned int count = 0;
    if (isnan(value[i])) {
      count = atomicAdd(&nan_count, 1);
    } else if (isinf(value[i])) {
      count = atomicAdd(&inf_count, 1);
    } else {
      count = atomicAdd(&num_count, 1);
    }
    // for cuda, print in every block
    if (count < print_num) {
      printf("numel:%lu idx:%lu value:%f\n", static_cast<uint64_t>(numel),
             static_cast<uint64_t>(i), static_cast<float>(value[i]));
    }
  }
  __syncthreads;

#ifdef PADDLE_WITH_HIP
  if (true && hipThreadIdx_x == 0) {
    printf("In block %d, there has %u,%u,%u nan,inf,num\n", hipBlockIdx_x,
           nan_count, inf_count, num_count);
#else
  if (true && threadIdx.x == 0) {
    printf("In block %d, there has %u,%u,%u nan,inf,num\n", blockIdx.x,
           nan_count, inf_count, num_count);
#endif
    PADDLE_ENFORCE(false, "===ERROR: in %s find nan or inf===", debug_info);
  }
}

// Resnet 2gpus speed test, no check 270 images/s, this check 229 images/s
template <typename T>
__global__ void CheckNanInfKernel(const T* value, const size_t numel,
                                  int print_num, char* debug_info) {
  /// step 1, judge wheater has nan or inf
  __shared__ volatile int has_nan_inf;
  if (threadIdx.x == 0) has_nan_inf = false;
  __syncthreads();

  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  T sum = static_cast<T>(0.0);
  // Todo(wangxi). simd speed up
  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    sum += (value[i] - value[i]);
  }

  if (isnan(sum) || isinf(sum)) has_nan_inf = true;
  __syncthreads();

  /// Note. different blocks may behave differently
  if (!has_nan_inf) return;

  PrintNanInfKernel(value, numel, print_num, debug_info);
}

template <>
template <typename T>
void TensorCheckerVisitor<platform::CUDADeviceContext>::apply(
    typename std::enable_if<std::is_floating_point<T>::value>::type*) const {
  int print_num = 3;

  auto* dev_ctx = reinterpret_cast<platform::CUDADeviceContext*>(
      platform::DeviceContextPool::Instance().Get(tensor_.place()));
  int dev_id = BOOST_GET_CONST(platform::CUDAPlace, tensor_.place()).device;
  PADDLE_ENFORCE_EQ(
      (dev_id >= 0 && dev_id < multi_op_var2gpu_str_mutex().size()), true,
      platform::errors::OutOfRange("GPU dev_id must >=0 and < dev_count=%d",
                                   multi_op_var2gpu_str_mutex().size()));

  std::string op_var = "[op=" + op_type_ + "] [tensor=" + var_name_ + "]";
  char* gpu_str_ptr = NULL;

  {
    auto& op_var2gpu_str_mutex = multi_op_var2gpu_str_mutex().at(dev_id);
    auto& op_var2gpu_str = multi_op_var2gpu_str().at(dev_id);

    std::lock_guard<std::mutex> guard(op_var2gpu_str_mutex);
    if (op_var2gpu_str.find(op_var) == op_var2gpu_str.end()) {  // insert
      auto gpu_str_tensor =
          paddle::memory::Alloc(*dev_ctx, op_var.length() + 1);
      gpu_str_ptr = reinterpret_cast<char*>(gpu_str_tensor->ptr());

      op_var2gpu_str.emplace(op_var, std::move(gpu_str_tensor));

      auto iter = op_var2gpu_str.find(op_var);
      PADDLE_ENFORCE_EQ(iter != op_var2gpu_str.end(), true,
                        platform::errors::PreconditionNotMet(
                            "op_var=%s should successed insert into "
                            "op_var2gpu_str, but now failed",
                            op_var));

#ifdef PADDLE_WITH_HIP
      PADDLE_ENFORCE_CUDA_SUCCESS(
          hipMemcpyAsync(gpu_str_ptr, iter->first.c_str(), op_var.length() + 1,
                         hipMemcpyHostToDevice, dev_ctx->stream()));
#else
      PADDLE_ENFORCE_CUDA_SUCCESS(
          hipMemcpyAsync(gpu_str_ptr, iter->first.c_str(), op_var.length() + 1,
                          hipMemcpyHostToDevice, dev_ctx->stream()));
#endif
    } else {  // get
      auto iter = op_var2gpu_str.find(op_var);
      PADDLE_ENFORCE_EQ(iter != op_var2gpu_str.end(), true,
                        platform::errors::PreconditionNotMet(
                            "op_var=%s should be in the op_var2gpu_str, but "
                            "now can't find it",
                            op_var));
      gpu_str_ptr = reinterpret_cast<char*>(iter->second->ptr());
    }
  }

  const size_t threads = 1024;
  size_t blocks =
      std::min(static_cast<size_t>(128),
               static_cast<size_t>((tensor_.numel() + threads - 1) / threads));
#ifdef PADDLE_WITH_HIP
  hipLaunchKernelGGL(CheckNanInfKernel, dim3(blocks), dim3(threads), 0,
                     dev_ctx->stream(), tensor_.data<T>(), tensor_.numel(),
                     print_num, gpu_str_ptr);
#else
  CheckNanInfKernel<<<blocks, threads, 0, dev_ctx->stream()>>>(
      tensor_.data<T>(), tensor_.numel(), print_num, gpu_str_ptr);
#endif
}

template <>
void tensor_check<platform::CUDADeviceContext>(const std::string& op_type,
                                               const std::string& var_name,
                                               const framework::Tensor& tensor,
                                               const platform::Place& place) {
  std::call_once(init_multi_gpu_op_var_map_flag, InitMultiGPUOpVarMap);

  TensorCheckerVisitor<platform::CUDADeviceContext> vistor(op_type, var_name,
                                                           tensor, place);
  VisitDataType(tensor.type(), vistor);
}

}  // namespace details
}  // namespace framework
}  // namespace paddle
