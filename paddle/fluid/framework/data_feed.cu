
#include <hip/hip_runtime.h>
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#if defined _WIN32 || defined __APPLE__
#else
#define _LINUX
#endif
#if defined(PADDLE_WITH_CUDA) && defined(PADDLE_WITH_HETERPS)

#include "paddle/fluid/framework/data_feed.h"
#include <thrust/device_ptr.h>
#include <thrust/random.h>
#include <thrust/shuffle.h>
#include <sstream>
#include "cub/cub.cuh"
#include "paddle/fluid/framework/fleet/heter_ps/gpu_graph_node.h"
<<<<<<< HEAD
#include "paddle/fluid/framework/fleet/heter_ps/gpu_graph_utils.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_wrapper.h"
#include "paddle/fluid/framework/fleet/heter_ps/hashtable.h"
#include "paddle/fluid/framework/fleet/ps_gpu_wrapper.h"

DECLARE_bool(enable_opt_get_features);
DECLARE_int32(gpugraph_storage_mode);
DECLARE_double(gpugraph_hbm_table_load_factor);
=======
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_wrapper.h"

DECLARE_bool(enable_opt_get_features);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f

namespace paddle {
namespace framework {

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

#define DEBUG_STATE(state)                                             \
  VLOG(2) << "left: " << state->left << " right: " << state->right     \
          << " central_word: " << state->central_word                  \
          << " step: " << state->step << " cursor: " << state->cursor  \
          << " len: " << state->len << " row_num: " << state->row_num; \
// CUDA: use 512 threads per block
const int CUDA_NUM_THREADS = 512;
// CUDA: number of blocks for threads.
inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
// fill slot values
__global__ void FillSlotValueOffsetKernel(const int ins_num,
                                          const int used_slot_num,
                                          size_t *slot_value_offsets,
                                          const int *uint64_offsets,
                                          const int uint64_slot_size,
                                          const int *float_offsets,
                                          const int float_slot_size,
                                          const UsedSlotGpuType *used_slots) {
  int col_num = ins_num + 1;
  int uint64_cols = uint64_slot_size + 1;
  int float_cols = float_slot_size + 1;

  CUDA_KERNEL_LOOP(slot_idx, used_slot_num) {
    int value_off = slot_idx * col_num;
    slot_value_offsets[value_off] = 0;

    auto &info = used_slots[slot_idx];
    if (info.is_uint64_value) {
      for (int k = 0; k < ins_num; ++k) {
        int pos = k * uint64_cols + info.slot_value_idx;
        int num = uint64_offsets[pos + 1] - uint64_offsets[pos];
        PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
        slot_value_offsets[value_off + k + 1] =
            slot_value_offsets[value_off + k] + num;
      }
    } else {
      for (int k = 0; k < ins_num; ++k) {
        int pos = k * float_cols + info.slot_value_idx;
        int num = float_offsets[pos + 1] - float_offsets[pos];
        PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
        slot_value_offsets[value_off + k + 1] =
            slot_value_offsets[value_off + k] + num;
      }
    }
  }
}

void SlotRecordInMemoryDataFeed::FillSlotValueOffset(
    const int ins_num,
    const int used_slot_num,
    size_t *slot_value_offsets,
    const int *uint64_offsets,
    const int uint64_slot_size,
    const int *float_offsets,
    const int float_slot_size,
    const UsedSlotGpuType *used_slots) {
  auto stream =
      dynamic_cast<phi::GPUContext *>(
          paddle::platform::DeviceContextPool::Instance().Get(this->place_))
          ->stream();
  FillSlotValueOffsetKernel<<<GET_BLOCKS(used_slot_num),
                              CUDA_NUM_THREADS,
                              0,
                              stream>>>(ins_num,
                                        used_slot_num,
                                        slot_value_offsets,
                                        uint64_offsets,
                                        uint64_slot_size,
                                        float_offsets,
                                        float_slot_size,
                                        used_slots);
  cudaStreamSynchronize(stream);
}

__global__ void CopyForTensorKernel(const int used_slot_num,
                                    const int ins_num,
                                    void **dest,
                                    const size_t *slot_value_offsets,
                                    const uint64_t *uint64_feas,
                                    const int *uint64_offsets,
                                    const int *uint64_ins_lens,
                                    const int uint64_slot_size,
                                    const float *float_feas,
                                    const int *float_offsets,
                                    const int *float_ins_lens,
                                    const int float_slot_size,
                                    const UsedSlotGpuType *used_slots) {
  int col_num = ins_num + 1;
  int uint64_cols = uint64_slot_size + 1;
  int float_cols = float_slot_size + 1;

  CUDA_KERNEL_LOOP(i, ins_num * used_slot_num) {
    int slot_idx = i / ins_num;
    int ins_idx = i % ins_num;

    uint32_t value_offset = slot_value_offsets[slot_idx * col_num + ins_idx];
    auto &info = used_slots[slot_idx];
    if (info.is_uint64_value) {
      uint64_t *up = reinterpret_cast<uint64_t *>(dest[slot_idx]);
      int index = info.slot_value_idx + uint64_cols * ins_idx;
      int old_off = uint64_offsets[index];
      int num = uint64_offsets[index + 1] - old_off;
      PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
      int uint64_value_offset = uint64_ins_lens[ins_idx];
      for (int k = 0; k < num; ++k) {
        up[k + value_offset] = uint64_feas[k + old_off + uint64_value_offset];
      }
    } else {
      float *fp = reinterpret_cast<float *>(dest[slot_idx]);
      int index = info.slot_value_idx + float_cols * ins_idx;
      int old_off = float_offsets[index];
      int num = float_offsets[index + 1] - old_off;
      PADDLE_ENFORCE(num >= 0, "The number of slot size must be ge 0.");
      int float_value_offset = float_ins_lens[ins_idx];
      for (int k = 0; k < num; ++k) {
        fp[k + value_offset] = float_feas[k + old_off + float_value_offset];
      }
    }
  }
}

void SlotRecordInMemoryDataFeed::CopyForTensor(
    const int ins_num,
    const int used_slot_num,
    void **dest,
    const size_t *slot_value_offsets,
    const uint64_t *uint64_feas,
    const int *uint64_offsets,
    const int *uint64_ins_lens,
    const int uint64_slot_size,
    const float *float_feas,
    const int *float_offsets,
    const int *float_ins_lens,
    const int float_slot_size,
    const UsedSlotGpuType *used_slots) {
  auto stream =
      dynamic_cast<phi::GPUContext *>(
          paddle::platform::DeviceContextPool::Instance().Get(this->place_))
          ->stream();

  CopyForTensorKernel<<<GET_BLOCKS(used_slot_num * ins_num),
                        CUDA_NUM_THREADS,
                        0,
                        stream>>>(used_slot_num,
                                  ins_num,
                                  dest,
                                  slot_value_offsets,
                                  uint64_feas,
                                  uint64_offsets,
                                  uint64_ins_lens,
                                  uint64_slot_size,
                                  float_feas,
                                  float_offsets,
                                  float_ins_lens,
                                  float_slot_size,
                                  used_slots);
  cudaStreamSynchronize(stream);
}

__global__ void GraphFillCVMKernel(int64_t *tensor, int len) {
  CUDA_KERNEL_LOOP(idx, len) { tensor[idx] = 1; }
}

__global__ void CopyDuplicateKeys(int64_t *dist_tensor,
                                  uint64_t *src_tensor,
                                  int len) {
  CUDA_KERNEL_LOOP(idx, len) {
    dist_tensor[idx * 2] = src_tensor[idx];
    dist_tensor[idx * 2 + 1] = src_tensor[idx];
  }
}

int GraphDataGenerator::AcquireInstance(BufState *state) {
  //
  if (state->GetNextStep()) {
<<<<<<< HEAD
    DEBUG_STATE(state);
    return state->len;
  } else if (state->GetNextCentrolWord()) {
    DEBUG_STATE(state);
    return state->len;
  } else if (state->GetNextBatch()) {
    DEBUG_STATE(state);
=======
    state->Debug();
    return state->len;
  } else if (state->GetNextCentrolWord()) {
    state->Debug();
    return state->len;
  } else if (state->GetNextBatch()) {
    state->Debug();
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    return state->len;
  }
  return 0;
}

<<<<<<< HEAD
// TODO opt
=======
// TODO(fengdanlei): opt
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
__global__ void GraphFillFeatureKernel(uint64_t *id_tensor,
                                       int *fill_ins_num,
                                       uint64_t *walk,
                                       uint64_t *feature,
                                       int *row,
                                       int central_word,
                                       int step,
                                       int len,
                                       int col_num,
                                       int slot_num) {
  __shared__ int32_t local_key[CUDA_NUM_THREADS * 16];
  __shared__ int local_num;
  __shared__ int global_num;

  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();
  if (idx < len) {
    int src = row[idx] * col_num + central_word;
    if (walk[src] != 0 && walk[src + step] != 0) {
      size_t dst = atomicAdd(&local_num, 1);
      for (int i = 0; i < slot_num; ++i) {
        local_key[dst * 2 * slot_num + i * 2] = feature[src * slot_num + i];
        local_key[dst * 2 * slot_num + i * 2 + 1] =
            feature[(src + step) * slot_num + i];
      }
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    global_num = atomicAdd(fill_ins_num, local_num);
  }
  __syncthreads();

  if (threadIdx.x < local_num) {
    for (int i = 0; i < slot_num; ++i) {
      id_tensor[(global_num * 2 + 2 * threadIdx.x) * slot_num + i] =
          local_key[(2 * threadIdx.x) * slot_num + i];
      id_tensor[(global_num * 2 + 2 * threadIdx.x + 1) * slot_num + i] =
          local_key[(2 * threadIdx.x + 1) * slot_num + i];
    }
  }
}

__global__ void GraphFillIdKernel(uint64_t *id_tensor,
                                  int *fill_ins_num,
                                  uint64_t *walk,
                                  int *row,
                                  int central_word,
                                  int step,
                                  int len,
                                  int col_num) {
  __shared__ uint64_t local_key[CUDA_NUM_THREADS * 2];
  __shared__ int local_num;
  __shared__ int global_num;

  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();
  // int dst = idx * 2;
  // id_tensor[dst] = walk[src];
  // id_tensor[dst + 1] = walk[src + step];
  if (idx < len) {
    int src = row[idx] * col_num + central_word;
    if (walk[src] != 0 && walk[src + step] != 0) {
      size_t dst = atomicAdd(&local_num, 1);
      local_key[dst * 2] = walk[src];
      local_key[dst * 2 + 1] = walk[src + step];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    global_num = atomicAdd(fill_ins_num, local_num);
  }
  __syncthreads();

  if (threadIdx.x < local_num) {
    id_tensor[global_num * 2 + 2 * threadIdx.x] = local_key[2 * threadIdx.x];
    id_tensor[global_num * 2 + 2 * threadIdx.x + 1] =
        local_key[2 * threadIdx.x + 1];
  }
}

__global__ void GraphFillSlotKernel(uint64_t *id_tensor,
                                    uint64_t *feature_buf,
                                    int len,
                                    int total_ins,
<<<<<<< HEAD
                                    int slot_num,
                                    int *slot_feature_num_map,
                                    int fea_num_per_node,
                                    int *actual_slot_id_map,
                                    int *fea_offset_map) {
  CUDA_KERNEL_LOOP(idx, len) {
    int fea_idx = idx / total_ins;
    int ins_idx = idx % total_ins;
    int actual_slot_id = actual_slot_id_map[fea_idx];
    int fea_offset = fea_offset_map[fea_idx];
    ((uint64_t *)(id_tensor[actual_slot_id]))
        [ins_idx * slot_feature_num_map[actual_slot_id] + fea_offset] =
            feature_buf[ins_idx * fea_num_per_node + fea_idx];
=======
                                    int slot_num) {
  CUDA_KERNEL_LOOP(idx, len) {
    int slot_idx = idx / total_ins;
    int ins_idx = idx % total_ins;
    ((uint64_t *)(id_tensor[slot_idx]))[ins_idx] =  // NOLINT
        feature_buf[ins_idx * slot_num + slot_idx];
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  }
}

__global__ void GraphFillSlotLodKernelOpt(uint64_t *id_tensor,
                                          int len,
<<<<<<< HEAD
                                          int total_ins,
                                          int *slot_feature_num_map) {
  CUDA_KERNEL_LOOP(idx, len) {
    int slot_idx = idx / total_ins;
    int ins_idx = idx % total_ins;
    ((uint64_t *)(id_tensor[slot_idx]))[ins_idx] =
        ins_idx * slot_feature_num_map[slot_idx];
=======
                                          int total_ins) {
  CUDA_KERNEL_LOOP(idx, len) {
    int slot_idx = idx / total_ins;
    int ins_idx = idx % total_ins;
    ((uint64_t *)(id_tensor[slot_idx]))[ins_idx] = ins_idx;  // NOLINT
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  }
}

__global__ void GraphFillSlotLodKernel(int64_t *id_tensor, int len) {
  CUDA_KERNEL_LOOP(idx, len) { id_tensor[idx] = idx; }
}

<<<<<<< HEAD
int GraphDataGenerator::FillIdShowClkTensor(int total_instance,
                                            bool gpu_graph_training,
                                            size_t cursor) {
  id_tensor_ptr_ =
      feed_vec_[0]->mutable_data<int64_t>({total_instance, 1}, this->place_);
  show_tensor_ptr_ =
      feed_vec_[1]->mutable_data<int64_t>({total_instance}, this->place_);
  clk_tensor_ptr_ =
      feed_vec_[2]->mutable_data<int64_t>({total_instance}, this->place_);
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  if (gpu_graph_training) {
    uint64_t *ins_cursor, *ins_buf;
    ins_buf = reinterpret_cast<uint64_t *>(d_ins_buf_->ptr());
    ins_cursor = ins_buf + ins_buf_pair_len_ * 2 - total_instance;
    cudaMemcpyAsync(id_tensor_ptr_,
                    ins_cursor,
                    sizeof(uint64_t) * total_instance,
                    cudaMemcpyDeviceToDevice,
                    train_stream_);
  } else {
    uint64_t *d_type_keys =
        reinterpret_cast<uint64_t *>(d_device_keys_[cursor]->ptr());
    d_type_keys += infer_node_start_;
    infer_node_start_ += total_instance / 2;
    CopyDuplicateKeys<<<GET_BLOCKS(total_instance / 2),
                        CUDA_NUM_THREADS,
                        0,
                        train_stream_>>>(
        id_tensor_ptr_, d_type_keys, total_instance / 2);
  }

  GraphFillCVMKernel<<<GET_BLOCKS(total_instance),
                       CUDA_NUM_THREADS,
                       0,
                       train_stream_>>>(show_tensor_ptr_, total_instance);
  GraphFillCVMKernel<<<GET_BLOCKS(total_instance),
                       CUDA_NUM_THREADS,
                       0,
                       train_stream_>>>(clk_tensor_ptr_, total_instance);
  return 0;
}

int GraphDataGenerator::FillGraphSlotFeature(int total_instance,
                                             bool gpu_graph_training) {
  uint64_t *ins_cursor, *ins_buf;
  if (gpu_graph_training) {
    ins_buf = reinterpret_cast<uint64_t *>(d_ins_buf_->ptr());
    ins_cursor = ins_buf + ins_buf_pair_len_ * 2 - total_instance;
  } else {
    id_tensor_ptr_ =
        feed_vec_[0]->mutable_data<int64_t>({total_instance, 1}, this->place_);
    ins_cursor = (uint64_t *)id_tensor_ptr_;
  }
  return FillSlotFeature(ins_cursor, total_instance);
}

int GraphDataGenerator::MakeInsPair() {
=======
int GraphDataGenerator::FillInsBuf() {
  if (ins_buf_pair_len_ >= batch_size_) {
    return batch_size_;
  }
  int total_instance = AcquireInstance(&buf_state_);

  VLOG(2) << "total_ins: " << total_instance;
  buf_state_.Debug();

  if (total_instance == 0) {
    int res = FillWalkBuf(d_walk_);
    if (!res) {
      // graph iterate complete
      return -1;
    } else {
      total_instance = buf_state_.len;
      VLOG(2) << "total_ins: " << total_instance;
      buf_state_.Debug();
      // if (total_instance == 0) {
      //  return -1;
      //}
    }

    if (!FLAGS_enable_opt_get_features && slot_num_ > 0) {
      FillFeatureBuf(d_walk_, d_feature_);
      if (debug_mode_) {
        int len = buf_size_ > 5000 ? 5000 : buf_size_;
        uint64_t h_walk[len];  // NOLINT
        cudaMemcpy(h_walk,
                   d_walk_->ptr(),
                   len * sizeof(uint64_t),
                   cudaMemcpyDeviceToHost);
        uint64_t h_feature[len * slot_num_];  // NOLINT
        cudaMemcpy(h_feature,
                   d_feature_->ptr(),
                   len * slot_num_ * sizeof(uint64_t),
                   cudaMemcpyDeviceToHost);
        for (int i = 0; i < len; ++i) {
          std::stringstream ss;
          for (int j = 0; j < slot_num_; ++j) {
            ss << h_feature[i * slot_num_ + j] << " ";
          }
          VLOG(2) << "aft FillFeatureBuf, gpu[" << gpuid_ << "] walk[" << i
                  << "] = " << (uint64_t)h_walk[i] << " feature["
                  << i * slot_num_ << ".." << (i + 1) * slot_num_
                  << "] = " << ss.str();
        }
      }
    }
  }

>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  uint64_t *walk = reinterpret_cast<uint64_t *>(d_walk_->ptr());
  uint64_t *ins_buf = reinterpret_cast<uint64_t *>(d_ins_buf_->ptr());
  int *random_row = reinterpret_cast<int *>(d_random_row_->ptr());
  int *d_pair_num = reinterpret_cast<int *>(d_pair_num_->ptr());
<<<<<<< HEAD
  cudaMemsetAsync(d_pair_num, 0, sizeof(int), train_stream_);
  int len = buf_state_.len;
  // make pair
  GraphFillIdKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, train_stream_>>>(
=======
  cudaMemsetAsync(d_pair_num, 0, sizeof(int), stream_);
  int len = buf_state_.len;
  GraphFillIdKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
      ins_buf + ins_buf_pair_len_ * 2,
      d_pair_num,
      walk,
      random_row + buf_state_.cursor,
      buf_state_.central_word,
      window_step_[buf_state_.step],
      len,
      walk_len_);
  int h_pair_num;
<<<<<<< HEAD
  cudaMemcpyAsync(&h_pair_num,
                  d_pair_num,
                  sizeof(int),
                  cudaMemcpyDeviceToHost,
                  train_stream_);
  cudaStreamSynchronize(train_stream_);
  ins_buf_pair_len_ += h_pair_num;

  if (debug_mode_) {
    uint64_t h_ins_buf[ins_buf_pair_len_ * 2];
=======
  cudaMemcpyAsync(
      &h_pair_num, d_pair_num, sizeof(int), cudaMemcpyDeviceToHost, stream_);
  if (!FLAGS_enable_opt_get_features && slot_num_ > 0) {
    uint64_t *feature_buf = reinterpret_cast<uint64_t *>(d_feature_buf_->ptr());
    uint64_t *feature = reinterpret_cast<uint64_t *>(d_feature_->ptr());
    cudaMemsetAsync(d_pair_num, 0, sizeof(int), stream_);
    int len = buf_state_.len;
    VLOG(2) << "feature_buf start[" << ins_buf_pair_len_ * 2 * slot_num_
            << "] len[" << len << "]";
    GraphFillFeatureKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(
        feature_buf + ins_buf_pair_len_ * 2 * slot_num_,
        d_pair_num,
        walk,
        feature,
        random_row + buf_state_.cursor,
        buf_state_.central_word,
        window_step_[buf_state_.step],
        len,
        walk_len_,
        slot_num_);
  }

  cudaStreamSynchronize(stream_);
  ins_buf_pair_len_ += h_pair_num;

  if (debug_mode_) {
    uint64_t h_ins_buf[ins_buf_pair_len_ * 2];  // NOLINT
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    cudaMemcpy(h_ins_buf,
               ins_buf,
               2 * ins_buf_pair_len_ * sizeof(uint64_t),
               cudaMemcpyDeviceToHost);
    VLOG(2) << "h_pair_num = " << h_pair_num
            << ", ins_buf_pair_len = " << ins_buf_pair_len_;
    for (int xx = 0; xx < 2 * ins_buf_pair_len_; xx++) {
      VLOG(2) << "h_ins_buf[" << xx << "]: " << h_ins_buf[xx];
    }
<<<<<<< HEAD
=======
    delete[] h_ins_buf;

    if (!FLAGS_enable_opt_get_features && slot_num_ > 0) {
      uint64_t *feature_buf =
          reinterpret_cast<uint64_t *>(d_feature_buf_->ptr());
      uint64_t h_feature_buf[(batch_size_ * 2 * 2) * slot_num_];  // NOLINT
      cudaMemcpy(h_feature_buf,
                 feature_buf,
                 (batch_size_ * 2 * 2) * slot_num_ * sizeof(uint64_t),
                 cudaMemcpyDeviceToHost);
      for (int xx = 0; xx < (batch_size_ * 2 * 2) * slot_num_; xx++) {
        VLOG(2) << "h_feature_buf[" << xx << "]: " << h_feature_buf[xx];
      }
    }
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  }
  return ins_buf_pair_len_;
}

<<<<<<< HEAD
int GraphDataGenerator::FillInsBuf() {
  if (ins_buf_pair_len_ >= batch_size_) {
    return batch_size_;
  }
  int total_instance = AcquireInstance(&buf_state_);

  VLOG(2) << "total_ins: " << total_instance;
  buf_state_.Debug();

  if (total_instance == 0) {
    return -1;
  }
  return MakeInsPair();
}

=======
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
int GraphDataGenerator::GenerateBatch() {
  int total_instance = 0;
  platform::CUDADeviceGuard guard(gpuid_);
  int res = 0;
<<<<<<< HEAD
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  if (!gpu_graph_training_) {
    total_instance = (infer_node_start_ + batch_size_ <= infer_node_end_)
                         ? batch_size_
                         : infer_node_end_ - infer_node_start_;
    VLOG(1) << "in graph_data generator:batch_size = " << batch_size_
            << " instance = " << total_instance;
    total_instance *= 2;
    if (total_instance == 0) {
      return 0;
    }
    FillIdShowClkTensor(total_instance, gpu_graph_training_, cursor_);
=======
  if (!gpu_graph_training_) {
    while (cursor_ < h_device_keys_.size()) {
      size_t device_key_size = h_device_keys_[cursor_]->size();
      if (infer_node_type_start_[cursor_] >= device_key_size) {
        cursor_++;
        continue;
      }
      total_instance =
          (infer_node_type_start_[cursor_] + batch_size_ <= device_key_size)
              ? batch_size_
              : device_key_size - infer_node_type_start_[cursor_];
      uint64_t *d_type_keys =
          reinterpret_cast<uint64_t *>(d_device_keys_[cursor_]->ptr());
      d_type_keys += infer_node_type_start_[cursor_];
      infer_node_type_start_[cursor_] += total_instance;
      VLOG(1) << "in graph_data generator:batch_size = " << batch_size_
              << " instance = " << total_instance;
      total_instance *= 2;
      id_tensor_ptr_ = feed_vec_[0]->mutable_data<int64_t>({total_instance, 1},
                                                           this->place_);
      show_tensor_ptr_ =
          feed_vec_[1]->mutable_data<int64_t>({total_instance}, this->place_);
      clk_tensor_ptr_ =
          feed_vec_[2]->mutable_data<int64_t>({total_instance}, this->place_);
      CopyDuplicateKeys<<<GET_BLOCKS(total_instance / 2),
                          CUDA_NUM_THREADS,
                          0,
                          stream_>>>(
          id_tensor_ptr_, d_type_keys, total_instance / 2);
      GraphFillCVMKernel<<<GET_BLOCKS(total_instance),
                           CUDA_NUM_THREADS,
                           0,
                           stream_>>>(show_tensor_ptr_, total_instance);
      GraphFillCVMKernel<<<GET_BLOCKS(total_instance),
                           CUDA_NUM_THREADS,
                           0,
                           stream_>>>(clk_tensor_ptr_, total_instance);
      break;
    }
    if (total_instance == 0) {
      return 0;
    }
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  } else {
    while (ins_buf_pair_len_ < batch_size_) {
      res = FillInsBuf();
      if (res == -1) {
        if (ins_buf_pair_len_ == 0) {
          return 0;
        } else {
          break;
        }
      }
    }
    total_instance =
        ins_buf_pair_len_ < batch_size_ ? ins_buf_pair_len_ : batch_size_;
<<<<<<< HEAD
    total_instance *= 2;
    VLOG(2) << "total_instance: " << total_instance
            << ", ins_buf_pair_len = " << ins_buf_pair_len_;
    FillIdShowClkTensor(total_instance, gpu_graph_training_);
  }

  if (slot_num_ > 0) {
    FillGraphSlotFeature(total_instance, gpu_graph_training_);
  }
=======

    total_instance *= 2;
    id_tensor_ptr_ =
        feed_vec_[0]->mutable_data<int64_t>({total_instance, 1}, this->place_);
    show_tensor_ptr_ =
        feed_vec_[1]->mutable_data<int64_t>({total_instance}, this->place_);
    clk_tensor_ptr_ =
        feed_vec_[2]->mutable_data<int64_t>({total_instance}, this->place_);
  }

  int64_t *slot_tensor_ptr_[slot_num_];
  int64_t *slot_lod_tensor_ptr_[slot_num_];
  if (slot_num_ > 0) {
    for (int i = 0; i < slot_num_; ++i) {
      slot_tensor_ptr_[i] = feed_vec_[3 + 2 * i]->mutable_data<int64_t>(
          {total_instance, 1}, this->place_);
      slot_lod_tensor_ptr_[i] = feed_vec_[3 + 2 * i + 1]->mutable_data<int64_t>(
          {total_instance + 1}, this->place_);
    }
    if (FLAGS_enable_opt_get_features || !gpu_graph_training_) {
      cudaMemcpyAsync(d_slot_tensor_ptr_->ptr(),
                      slot_tensor_ptr_,
                      sizeof(uint64_t *) * slot_num_,
                      cudaMemcpyHostToDevice,
                      stream_);
      cudaMemcpyAsync(d_slot_lod_tensor_ptr_->ptr(),
                      slot_lod_tensor_ptr_,
                      sizeof(uint64_t *) * slot_num_,
                      cudaMemcpyHostToDevice,
                      stream_);
    }
  }

  uint64_t *ins_cursor, *ins_buf;
  if (gpu_graph_training_) {
    VLOG(2) << "total_instance: " << total_instance
            << ", ins_buf_pair_len = " << ins_buf_pair_len_;
    // uint64_t *ins_buf = reinterpret_cast<uint64_t *>(d_ins_buf_->ptr());
    // uint64_t *ins_cursor = ins_buf + ins_buf_pair_len_ * 2 - total_instance;
    ins_buf = reinterpret_cast<uint64_t *>(d_ins_buf_->ptr());
    ins_cursor = ins_buf + ins_buf_pair_len_ * 2 - total_instance;
    cudaMemcpyAsync(id_tensor_ptr_,
                    ins_cursor,
                    sizeof(uint64_t) * total_instance,
                    cudaMemcpyDeviceToDevice,
                    stream_);

    GraphFillCVMKernel<<<GET_BLOCKS(total_instance),
                         CUDA_NUM_THREADS,
                         0,
                         stream_>>>(show_tensor_ptr_, total_instance);
    GraphFillCVMKernel<<<GET_BLOCKS(total_instance),
                         CUDA_NUM_THREADS,
                         0,
                         stream_>>>(clk_tensor_ptr_, total_instance);
  } else {
    ins_cursor = (uint64_t *)id_tensor_ptr_;  // NOLINT
  }

  if (slot_num_ > 0) {
    uint64_t *feature_buf = reinterpret_cast<uint64_t *>(d_feature_buf_->ptr());
    if (FLAGS_enable_opt_get_features || !gpu_graph_training_) {
      FillFeatureBuf(ins_cursor, feature_buf, total_instance);
      // FillFeatureBuf(id_tensor_ptr_, feature_buf, total_instance);
      if (debug_mode_) {
        uint64_t h_walk[total_instance];  // NOLINT
        cudaMemcpy(h_walk,
                   ins_cursor,
                   total_instance * sizeof(uint64_t),
                   cudaMemcpyDeviceToHost);
        uint64_t h_feature[total_instance * slot_num_];  // NOLINT
        cudaMemcpy(h_feature,
                   feature_buf,
                   total_instance * slot_num_ * sizeof(uint64_t),
                   cudaMemcpyDeviceToHost);
        for (int i = 0; i < total_instance; ++i) {
          std::stringstream ss;
          for (int j = 0; j < slot_num_; ++j) {
            ss << h_feature[i * slot_num_ + j] << " ";
          }
          VLOG(2) << "aft FillFeatureBuf, gpu[" << gpuid_ << "] walk[" << i
                  << "] = " << (uint64_t)h_walk[i] << " feature["
                  << i * slot_num_ << ".." << (i + 1) * slot_num_
                  << "] = " << ss.str();
        }
      }

      GraphFillSlotKernel<<<GET_BLOCKS(total_instance * slot_num_),
                            CUDA_NUM_THREADS,
                            0,
                            stream_>>>(
          (uint64_t *)d_slot_tensor_ptr_->ptr(),  // NOLINT
          feature_buf,
          total_instance * slot_num_,
          total_instance,
          slot_num_);
      GraphFillSlotLodKernelOpt<<<GET_BLOCKS((total_instance + 1) * slot_num_),
                                  CUDA_NUM_THREADS,
                                  0,
                                  stream_>>>(
          (uint64_t *)d_slot_lod_tensor_ptr_->ptr(),  // NOLINT
          (total_instance + 1) * slot_num_,
          total_instance + 1);
    } else {
      for (int i = 0; i < slot_num_; ++i) {
        int feature_buf_offset =
            (ins_buf_pair_len_ * 2 - total_instance) * slot_num_ + i * 2;
        for (int j = 0; j < total_instance; j += 2) {
          VLOG(2) << "slot_tensor[" << i << "][" << j << "] <- feature_buf["
                  << feature_buf_offset + j * slot_num_ << "]";
          VLOG(2) << "slot_tensor[" << i << "][" << j + 1 << "] <- feature_buf["
                  << feature_buf_offset + j * slot_num_ + 1 << "]";
          cudaMemcpyAsync(slot_tensor_ptr_[i] + j,
                          &feature_buf[feature_buf_offset + j * slot_num_],
                          sizeof(uint64_t) * 2,
                          cudaMemcpyDeviceToDevice,
                          stream_);
        }
        GraphFillSlotLodKernel<<<GET_BLOCKS(total_instance),
                                 CUDA_NUM_THREADS,
                                 0,
                                 stream_>>>(slot_lod_tensor_ptr_[i],
                                            total_instance + 1);
      }
    }
  }

>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  offset_.clear();
  offset_.push_back(0);
  offset_.push_back(total_instance);
  LoD lod{offset_};
  feed_vec_[0]->set_lod(lod);
  if (slot_num_ > 0) {
    for (int i = 0; i < slot_num_; ++i) {
      feed_vec_[3 + 2 * i]->set_lod(lod);
    }
  }

<<<<<<< HEAD
  cudaStreamSynchronize(train_stream_);
  if (!gpu_graph_training_) return 1;
  ins_buf_pair_len_ -= total_instance / 2;
=======
  cudaStreamSynchronize(stream_);
  if (!gpu_graph_training_) return 1;
  ins_buf_pair_len_ -= total_instance / 2;
  if (debug_mode_) {
    uint64_t h_slot_tensor[slot_num_][total_instance];
    uint64_t h_slot_lod_tensor[slot_num_][total_instance + 1];
    for (int i = 0; i < slot_num_; ++i) {
      cudaMemcpy(h_slot_tensor[i],
                 slot_tensor_ptr_[i],
                 total_instance * sizeof(uint64_t),
                 cudaMemcpyDeviceToHost);
      int len = total_instance > 5000 ? 5000 : total_instance;
      for (int j = 0; j < len; ++j) {
        VLOG(2) << "gpu[" << gpuid_ << "] slot_tensor[" << i << "][" << j
                << "] = " << h_slot_tensor[i][j];
      }

      cudaMemcpy(h_slot_lod_tensor[i],
                 slot_lod_tensor_ptr_[i],
                 (total_instance + 1) * sizeof(uint64_t),
                 cudaMemcpyDeviceToHost);
      len = total_instance + 1 > 5000 ? 5000 : total_instance + 1;
      for (int j = 0; j < len; ++j) {
        VLOG(2) << "gpu[" << gpuid_ << "] slot_lod_tensor[" << i << "][" << j
                << "] = " << h_slot_lod_tensor[i][j];
      }
    }
  }

>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  return 1;
}

__global__ void GraphFillSampleKeysKernel(uint64_t *neighbors,
                                          uint64_t *sample_keys,
                                          int *prefix_sum,
                                          int *sampleidx2row,
                                          int *tmp_sampleidx2row,
                                          int *actual_sample_size,
                                          int cur_degree,
                                          int len) {
  CUDA_KERNEL_LOOP(idx, len) {
    for (int k = 0; k < actual_sample_size[idx]; k++) {
      size_t offset = prefix_sum[idx] + k;
      sample_keys[offset] = neighbors[idx * cur_degree + k];
      tmp_sampleidx2row[offset] = sampleidx2row[idx] + k;
    }
  }
}

__global__ void GraphDoWalkKernel(uint64_t *neighbors,
                                  uint64_t *walk,
                                  int *d_prefix_sum,
                                  int *actual_sample_size,
                                  int cur_degree,
                                  int step,
                                  int len,
                                  int *id_cnt,
                                  int *sampleidx2row,
                                  int col_size) {
  CUDA_KERNEL_LOOP(i, len) {
    for (int k = 0; k < actual_sample_size[i]; k++) {
      // int idx = sampleidx2row[i];
      size_t row = sampleidx2row[k + d_prefix_sum[i]];
      // size_t row = idx * cur_degree + k;
      size_t col = step;
      size_t offset = (row * col_size + col);
      walk[offset] = neighbors[i * cur_degree + k];
    }
  }
}

// Fill keys to the first column of walk
__global__ void GraphFillFirstStepKernel(int *prefix_sum,
                                         int *sampleidx2row,
                                         uint64_t *walk,
                                         uint64_t *keys,
                                         int len,
                                         int walk_degree,
                                         int col_size,
                                         int *actual_sample_size,
                                         uint64_t *neighbors,
                                         uint64_t *sample_keys) {
  CUDA_KERNEL_LOOP(idx, len) {
    for (int k = 0; k < actual_sample_size[idx]; k++) {
      size_t row = prefix_sum[idx] + k;
      sample_keys[row] = neighbors[idx * walk_degree + k];
      sampleidx2row[row] = row;

      size_t offset = col_size * row;
      walk[offset] = keys[idx];
      walk[offset + 1] = neighbors[idx * walk_degree + k];
    }
  }
}

<<<<<<< HEAD
__global__ void get_each_ins_info(uint8_t *slot_list,
                                  uint32_t *slot_size_list,
                                  uint32_t *slot_size_prefix,
                                  uint32_t *each_ins_slot_num,
                                  uint32_t *each_ins_slot_num_inner_prefix,
                                  size_t key_num,
                                  int slot_num) {
  const size_t i = blockIdx.x * blockDim.y + threadIdx.y;
  if (i < key_num) {
    uint32_t slot_index = slot_size_prefix[i];
    size_t each_ins_slot_index = i * slot_num;
    for (int j = 0; j < slot_size_list[i]; j++) {
      each_ins_slot_num[each_ins_slot_index + slot_list[slot_index + j]] += 1;
    }
    each_ins_slot_num_inner_prefix[each_ins_slot_index] = 0;
    for (int j = 1; j < slot_num; j++) {
      each_ins_slot_num_inner_prefix[each_ins_slot_index + j] =
          each_ins_slot_num[each_ins_slot_index + j - 1] +
          each_ins_slot_num_inner_prefix[each_ins_slot_index + j - 1];
    }
  }
}

__global__ void fill_slot_num(uint32_t *d_each_ins_slot_num_ptr,
                              uint64_t **d_ins_slot_num_vector_ptr,
                              size_t key_num,
                              int slot_num) {
  const size_t i = blockIdx.x * blockDim.y + threadIdx.y;
  if (i < key_num) {
    size_t d_each_index = i * slot_num;
    for (int j = 0; j < slot_num; j++) {
      d_ins_slot_num_vector_ptr[j][i] =
          d_each_ins_slot_num_ptr[d_each_index + j];
    }
  }
}

__global__ void fill_slot_tensor(uint64_t *feature_list,
                                 uint32_t *feature_size_prefixsum,
                                 uint32_t *each_ins_slot_num_inner_prefix,
                                 uint64_t *ins_slot_num,
                                 int64_t *slot_lod_tensor,
                                 int64_t *slot_tensor,
                                 int slot,
                                 int slot_num,
                                 size_t node_num) {
  const size_t i = blockIdx.x * blockDim.y + threadIdx.y;
  if (i < node_num) {
    size_t dst_index = slot_lod_tensor[i];
    size_t src_index = feature_size_prefixsum[i] +
                       each_ins_slot_num_inner_prefix[slot_num * i + slot];
    for (uint64_t j = 0; j < ins_slot_num[i]; j++) {
      slot_tensor[dst_index + j] = feature_list[src_index + j];
    }
  }
}

__global__ void GetUniqueFeaNum(uint64_t *d_in,
                                uint64_t *unique_num,
                                size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ uint64_t local_num;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();

  if (i < len - 1) {
    if (d_in[i] != d_in[i + 1]) {
      atomicAdd(&local_num, 1);
    }
  }
  if (i == len - 1) {
    atomicAdd(&local_num, 1);
  }

  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(unique_num, local_num);
  }
}

__global__ void UniqueFeature(uint64_t *d_in,
                              uint64_t *d_out,
                              uint64_t *unique_num,
                              size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ uint64_t local_key[CUDA_NUM_THREADS];
  __shared__ uint64_t local_num;
  __shared__ uint64_t global_num;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();

  if (i < len - 1) {
    if (d_in[i] != d_in[i + 1]) {
      size_t dst = atomicAdd(&local_num, 1);
      local_key[dst] = d_in[i];
    }
  }
  if (i == len - 1) {
    size_t dst = atomicAdd(&local_num, 1);
    local_key[dst] = d_in[i];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    global_num = atomicAdd(unique_num, local_num);
  }
  __syncthreads();

  if (threadIdx.x < local_num) {
    d_out[global_num + threadIdx.x] = local_key[threadIdx.x];
  }
}
=======
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
// Fill sample_res to the stepth column of walk
void GraphDataGenerator::FillOneStep(uint64_t *d_start_ids,
                                     uint64_t *walk,
                                     int len,
                                     NeighborSampleResult &sample_res,
                                     int cur_degree,
                                     int step,
                                     int *len_per_row) {
  size_t temp_storage_bytes = 0;
  int *d_actual_sample_size = sample_res.actual_sample_size;
  uint64_t *d_neighbors = sample_res.val;
  int *d_prefix_sum = reinterpret_cast<int *>(d_prefix_sum_->ptr());
  uint64_t *d_sample_keys = reinterpret_cast<uint64_t *>(d_sample_keys_->ptr());
  int *d_sampleidx2row =
      reinterpret_cast<int *>(d_sampleidx2rows_[cur_sampleidx2row_]->ptr());
  int *d_tmp_sampleidx2row =
      reinterpret_cast<int *>(d_sampleidx2rows_[1 - cur_sampleidx2row_]->ptr());

  CUDA_CHECK(cub::DeviceScan::InclusiveSum(NULL,
                                           temp_storage_bytes,
                                           d_actual_sample_size,
                                           d_prefix_sum + 1,
                                           len,
<<<<<<< HEAD
                                           sample_stream_));
  auto d_temp_storage = memory::Alloc(
      place_,
      temp_storage_bytes,
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
=======
                                           stream_));
  auto d_temp_storage = memory::Alloc(place_, temp_storage_bytes);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f

  CUDA_CHECK(cub::DeviceScan::InclusiveSum(d_temp_storage->ptr(),
                                           temp_storage_bytes,
                                           d_actual_sample_size,
                                           d_prefix_sum + 1,
                                           len,
<<<<<<< HEAD
                                           sample_stream_));

  cudaStreamSynchronize(sample_stream_);

  if (step == 1) {
    GraphFillFirstStepKernel<<<GET_BLOCKS(len),
                               CUDA_NUM_THREADS,
                               0,
                               sample_stream_>>>(d_prefix_sum,
                                                 d_tmp_sampleidx2row,
                                                 walk,
                                                 d_start_ids,
                                                 len,
                                                 walk_degree_,
                                                 walk_len_,
                                                 d_actual_sample_size,
                                                 d_neighbors,
                                                 d_sample_keys);
=======
                                           stream_));

  cudaStreamSynchronize(stream_);

  if (step == 1) {
    GraphFillFirstStepKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(
        d_prefix_sum,
        d_tmp_sampleidx2row,
        walk,
        d_start_ids,
        len,
        walk_degree_,
        walk_len_,
        d_actual_sample_size,
        d_neighbors,
        d_sample_keys);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f

  } else {
    GraphFillSampleKeysKernel<<<GET_BLOCKS(len),
                                CUDA_NUM_THREADS,
                                0,
<<<<<<< HEAD
                                sample_stream_>>>(d_neighbors,
                                                  d_sample_keys,
                                                  d_prefix_sum,
                                                  d_sampleidx2row,
                                                  d_tmp_sampleidx2row,
                                                  d_actual_sample_size,
                                                  cur_degree,
                                                  len);

    GraphDoWalkKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, sample_stream_>>>(
=======
                                stream_>>>(d_neighbors,
                                           d_sample_keys,
                                           d_prefix_sum,
                                           d_sampleidx2row,
                                           d_tmp_sampleidx2row,
                                           d_actual_sample_size,
                                           cur_degree,
                                           len);

    GraphDoWalkKernel<<<GET_BLOCKS(len), CUDA_NUM_THREADS, 0, stream_>>>(
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
        d_neighbors,
        walk,
        d_prefix_sum,
        d_actual_sample_size,
        cur_degree,
        step,
        len,
        len_per_row,
        d_tmp_sampleidx2row,
        walk_len_);
  }
  if (debug_mode_) {
    size_t once_max_sample_keynum = walk_degree_ * once_sample_startid_len_;
    int *h_prefix_sum = new int[len + 1];
    int *h_actual_size = new int[len];
    int *h_offset2idx = new int[once_max_sample_keynum];
<<<<<<< HEAD
=======
    uint64_t h_sample_keys[once_max_sample_keynum];  // NOLINT
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    cudaMemcpy(h_offset2idx,
               d_tmp_sampleidx2row,
               once_max_sample_keynum * sizeof(int),
               cudaMemcpyDeviceToHost);

    cudaMemcpy(h_prefix_sum,
               d_prefix_sum,
               (len + 1) * sizeof(int),
               cudaMemcpyDeviceToHost);
    for (int xx = 0; xx < once_max_sample_keynum; xx++) {
      VLOG(2) << "h_offset2idx[" << xx << "]: " << h_offset2idx[xx];
    }
    for (int xx = 0; xx < len + 1; xx++) {
      VLOG(2) << "h_prefix_sum[" << xx << "]: " << h_prefix_sum[xx];
    }
    delete[] h_prefix_sum;
    delete[] h_actual_size;
    delete[] h_offset2idx;
<<<<<<< HEAD
  }
  cudaStreamSynchronize(sample_stream_);
  cur_sampleidx2row_ = 1 - cur_sampleidx2row_;
}

int GraphDataGenerator::FillSlotFeature(uint64_t *d_walk, size_t key_num) {
  platform::CUDADeviceGuard guard(gpuid_);
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  std::shared_ptr<phi::Allocation> d_feature_list;
  std::shared_ptr<phi::Allocation> d_slot_list;
  uint32_t *d_feature_size_list_ptr =
      reinterpret_cast<uint32_t *>(d_feature_size_list_buf_->ptr());
  uint32_t *d_feature_size_prefixsum_ptr =
      reinterpret_cast<uint32_t *>(d_feature_size_prefixsum_buf_->ptr());
  int fea_num =
      gpu_graph_ptr->get_feature_info_of_nodes(gpuid_,
                                               d_walk,
                                               key_num,
                                               d_feature_size_list_ptr,
                                               d_feature_size_prefixsum_ptr,
                                               d_feature_list,
                                               d_slot_list);
  int64_t *slot_tensor_ptr_[slot_num_];
  int64_t *slot_lod_tensor_ptr_[slot_num_];
  if (fea_num == 0) {
    int64_t default_lod = 1;
    for (int i = 0; i < slot_num_; ++i) {
      slot_lod_tensor_ptr_[i] = feed_vec_[3 + 2 * i + 1]->mutable_data<int64_t>(
          {(long)key_num + 1}, this->place_);
      slot_tensor_ptr_[i] =
          feed_vec_[3 + 2 * i]->mutable_data<int64_t>({1, 1}, this->place_);
      CUDA_CHECK(cudaMemsetAsync(
          slot_tensor_ptr_[i], 0, sizeof(int64_t), train_stream_));
      CUDA_CHECK(cudaMemsetAsync(slot_lod_tensor_ptr_[i],
                                 0,
                                 sizeof(int64_t) * key_num,
                                 train_stream_));
      CUDA_CHECK(cudaMemcpyAsync(
          reinterpret_cast<char *>(slot_lod_tensor_ptr_[i] + key_num),
          &default_lod,
          sizeof(int64_t),
          cudaMemcpyHostToDevice,
          train_stream_));
    }
    CUDA_CHECK(cudaStreamSynchronize(train_stream_));
    return 0;
  }

  uint64_t *d_feature_list_ptr =
      reinterpret_cast<uint64_t *>(d_feature_list->ptr());
  uint8_t *d_slot_list_ptr = reinterpret_cast<uint8_t *>(d_slot_list->ptr());

  std::shared_ptr<phi::Allocation> d_each_ins_slot_num_inner_prefix =
      memory::AllocShared(place_, (slot_num_ * key_num) * sizeof(uint32_t));
  std::shared_ptr<phi::Allocation> d_each_ins_slot_num =
      memory::AllocShared(place_, (slot_num_ * key_num) * sizeof(uint32_t));
  uint32_t *d_each_ins_slot_num_ptr =
      reinterpret_cast<uint32_t *>(d_each_ins_slot_num->ptr());
  uint32_t *d_each_ins_slot_num_inner_prefix_ptr =
      reinterpret_cast<uint32_t *>(d_each_ins_slot_num_inner_prefix->ptr());
  CUDA_CHECK(cudaMemsetAsync(d_each_ins_slot_num_ptr,
                             0,
                             slot_num_ * key_num * sizeof(uint32_t),
                             train_stream_));

  dim3 grid((key_num - 1) / 256 + 1);
  dim3 block(1, 256);

  get_each_ins_info<<<grid, block, 0, train_stream_>>>(
      d_slot_list_ptr,
      d_feature_size_list_ptr,
      d_feature_size_prefixsum_ptr,
      d_each_ins_slot_num_ptr,
      d_each_ins_slot_num_inner_prefix_ptr,
      key_num,
      slot_num_);

  std::vector<std::shared_ptr<phi::Allocation>> ins_slot_num(slot_num_,
                                                             nullptr);
  std::vector<uint64_t *> ins_slot_num_vecotr(slot_num_, NULL);
  std::shared_ptr<phi::Allocation> d_ins_slot_num_vector =
      memory::AllocShared(place_, (slot_num_) * sizeof(uint64_t *));
  uint64_t **d_ins_slot_num_vector_ptr =
      reinterpret_cast<uint64_t **>(d_ins_slot_num_vector->ptr());
  for (int i = 0; i < slot_num_; i++) {
    ins_slot_num[i] = memory::AllocShared(place_, key_num * sizeof(uint64_t));
    ins_slot_num_vecotr[i] =
        reinterpret_cast<uint64_t *>(ins_slot_num[i]->ptr());
  }
  CUDA_CHECK(
      cudaMemcpyAsync(reinterpret_cast<char *>(d_ins_slot_num_vector_ptr),
                      ins_slot_num_vecotr.data(),
                      sizeof(uint64_t *) * slot_num_,
                      cudaMemcpyHostToDevice,
                      train_stream_));
  fill_slot_num<<<grid, block, 0, train_stream_>>>(
      d_each_ins_slot_num_ptr, d_ins_slot_num_vector_ptr, key_num, slot_num_);
  CUDA_CHECK(cudaStreamSynchronize(train_stream_));

  for (int i = 0; i < slot_num_; ++i) {
    slot_lod_tensor_ptr_[i] = feed_vec_[3 + 2 * i + 1]->mutable_data<int64_t>(
        {(long)key_num + 1}, this->place_);
  }
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(cub::DeviceScan::InclusiveSum(NULL,
                                           temp_storage_bytes,
                                           ins_slot_num_vecotr[0],
                                           slot_lod_tensor_ptr_[0] + 1,
                                           key_num,
                                           train_stream_));
  CUDA_CHECK(cudaStreamSynchronize(train_stream_));
  auto d_temp_storage = memory::Alloc(
      this->place_,
      temp_storage_bytes,
      phi::Stream(reinterpret_cast<phi::StreamId>(train_stream_)));
  std::vector<int64_t> each_slot_fea_num(slot_num_, 0);
  for (int i = 0; i < slot_num_; ++i) {
    CUDA_CHECK(cudaMemsetAsync(
        slot_lod_tensor_ptr_[i], 0, sizeof(uint64_t), train_stream_));
    CUDA_CHECK(cub::DeviceScan::InclusiveSum(d_temp_storage->ptr(),
                                             temp_storage_bytes,
                                             ins_slot_num_vecotr[i],
                                             slot_lod_tensor_ptr_[i] + 1,
                                             key_num,
                                             train_stream_));
    CUDA_CHECK(cudaMemcpyAsync(&each_slot_fea_num[i],
                               slot_lod_tensor_ptr_[i] + key_num,
                               sizeof(uint64_t),
                               cudaMemcpyDeviceToHost,
                               train_stream_));
  }
  CUDA_CHECK(cudaStreamSynchronize(train_stream_));
  for (int i = 0; i < slot_num_; ++i) {
    slot_tensor_ptr_[i] = feed_vec_[3 + 2 * i]->mutable_data<int64_t>(
        {each_slot_fea_num[i], 1}, this->place_);
  }
  int64_t default_lod = 1;
  for (int i = 0; i < slot_num_; ++i) {
    fill_slot_tensor<<<grid, block, 0, train_stream_>>>(
        d_feature_list_ptr,
        d_feature_size_prefixsum_ptr,
        d_each_ins_slot_num_inner_prefix_ptr,
        ins_slot_num_vecotr[i],
        slot_lod_tensor_ptr_[i],
        slot_tensor_ptr_[i],
        i,
        slot_num_,
        key_num);
    // trick for empty tensor
    if (each_slot_fea_num[i] == 0) {
      slot_tensor_ptr_[i] =
          feed_vec_[3 + 2 * i]->mutable_data<int64_t>({1, 1}, this->place_);
      CUDA_CHECK(cudaMemsetAsync(
          slot_tensor_ptr_[i], 0, sizeof(uint64_t), train_stream_));
      CUDA_CHECK(cudaMemcpyAsync(
          reinterpret_cast<char *>(slot_lod_tensor_ptr_[i] + key_num),
          &default_lod,
          sizeof(int64_t),
          cudaMemcpyHostToDevice,
          train_stream_));
    }
  }
  CUDA_CHECK(cudaStreamSynchronize(train_stream_));

  if (debug_mode_) {
    std::vector<uint32_t> h_feature_size_list(key_num, 0);
    std::vector<uint32_t> h_feature_size_list_prefixsum(key_num, 0);
    std::vector<uint64_t> node_list(key_num, 0);
    std::vector<uint64_t> h_feature_list(fea_num, 0);
    std::vector<uint8_t> h_slot_list(fea_num, 0);

    CUDA_CHECK(
        cudaMemcpyAsync(reinterpret_cast<char *>(h_feature_size_list.data()),
                        d_feature_size_list_ptr,
                        sizeof(uint32_t) * key_num,
                        cudaMemcpyDeviceToHost,
                        train_stream_));
    CUDA_CHECK(cudaMemcpyAsync(
        reinterpret_cast<char *>(h_feature_size_list_prefixsum.data()),
        d_feature_size_prefixsum_ptr,
        sizeof(uint32_t) * key_num,
        cudaMemcpyDeviceToHost,
        train_stream_));
    CUDA_CHECK(cudaMemcpyAsync(reinterpret_cast<char *>(node_list.data()),
                               d_walk,
                               sizeof(uint64_t) * key_num,
                               cudaMemcpyDeviceToHost,
                               train_stream_));

    CUDA_CHECK(cudaMemcpyAsync(reinterpret_cast<char *>(h_feature_list.data()),
                               d_feature_list_ptr,
                               sizeof(uint64_t) * fea_num,
                               cudaMemcpyDeviceToHost,
                               train_stream_));
    CUDA_CHECK(cudaMemcpyAsync(reinterpret_cast<char *>(h_slot_list.data()),
                               d_slot_list_ptr,
                               sizeof(uint8_t) * fea_num,
                               cudaMemcpyDeviceToHost,
                               train_stream_));

    CUDA_CHECK(cudaStreamSynchronize(train_stream_));
    for (size_t i = 0; i < key_num; i++) {
      std::stringstream ss;
      ss << "node_id: " << node_list[i]
         << " fea_num: " << h_feature_size_list[i] << " offset "
         << h_feature_size_list_prefixsum[i] << " slot: ";
      for (uint32_t j = 0; j < h_feature_size_list[i]; j++) {
        ss << int(h_slot_list[h_feature_size_list_prefixsum[i] + j]) << " : "
           << h_feature_list[h_feature_size_list_prefixsum[i] + j] << "  ";
      }
      VLOG(0) << ss.str();
    }
    VLOG(0) << "all fea_num is " << fea_num << " calc fea_num is "
            << h_feature_size_list[key_num - 1] +
                   h_feature_size_list_prefixsum[key_num - 1];
    for (int i = 0; i < slot_num_; ++i) {
      std::vector<int64_t> h_slot_lod_tensor(key_num + 1, 0);
      CUDA_CHECK(
          cudaMemcpyAsync(reinterpret_cast<char *>(h_slot_lod_tensor.data()),
                          slot_lod_tensor_ptr_[i],
                          sizeof(int64_t) * (key_num + 1),
                          cudaMemcpyDeviceToHost,
                          train_stream_));
      CUDA_CHECK(cudaStreamSynchronize(train_stream_));
      std::stringstream ss_lod;
      std::stringstream ss_tensor;
      ss_lod << " slot " << i << " lod is [";
      for (size_t j = 0; j < key_num + 1; j++) {
        ss_lod << h_slot_lod_tensor[j] << ",";
      }
      ss_lod << "]";
      std::vector<int64_t> h_slot_tensor(h_slot_lod_tensor[key_num], 0);
      CUDA_CHECK(cudaMemcpyAsync(reinterpret_cast<char *>(h_slot_tensor.data()),
                                 slot_tensor_ptr_[i],
                                 sizeof(int64_t) * h_slot_lod_tensor[key_num],
                                 cudaMemcpyDeviceToHost,
                                 train_stream_));
      CUDA_CHECK(cudaStreamSynchronize(train_stream_));

      ss_tensor << " tensor is [ ";
      for (size_t j = 0; j < h_slot_lod_tensor[key_num]; j++) {
        ss_tensor << h_slot_tensor[j] << ",";
      }
      ss_tensor << "]";
      VLOG(0) << ss_lod.str() << "  " << ss_tensor.str();
    }
  }

  return 0;
}

=======
    delete[] h_sample_keys;
  }
  cudaStreamSynchronize(stream_);
  cur_sampleidx2row_ = 1 - cur_sampleidx2row_;
}

>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
int GraphDataGenerator::FillFeatureBuf(uint64_t *d_walk,
                                       uint64_t *d_feature,
                                       size_t key_num) {
  platform::CUDADeviceGuard guard(gpuid_);

  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
<<<<<<< HEAD
  int ret =
      gpu_graph_ptr->get_feature_of_nodes(gpuid_,
                                          d_walk,
                                          d_feature,
                                          key_num,
                                          slot_num_,
                                          (int *)d_slot_feature_num_map_->ptr(),
                                          fea_num_per_node_);
=======
  int ret = gpu_graph_ptr->get_feature_of_nodes(
      gpuid_, d_walk, d_feature, key_num, slot_num_);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  return ret;
}

int GraphDataGenerator::FillFeatureBuf(
    std::shared_ptr<phi::Allocation> d_walk,
    std::shared_ptr<phi::Allocation> d_feature) {
  platform::CUDADeviceGuard guard(gpuid_);

  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
<<<<<<< HEAD
  int ret =
      gpu_graph_ptr->get_feature_of_nodes(gpuid_,
                                          (uint64_t *)d_walk->ptr(),
                                          (uint64_t *)d_feature->ptr(),
                                          buf_size_,
                                          slot_num_,
                                          (int *)d_slot_feature_num_map_->ptr(),
                                          fea_num_per_node_);
  return ret;
}

// 尝试插入table, 0表示插入成功
int GraphDataGenerator::InsertTable(
    const unsigned long *d_keys,
    unsigned long len,
    std::shared_ptr<phi::Allocation> d_uniq_node_num) {
  uint64_t h_uniq_node_num = 0;
  uint64_t *d_uniq_node_num_ptr =
      reinterpret_cast<uint64_t *>(d_uniq_node_num->ptr());
  cudaMemcpyAsync(&h_uniq_node_num,
                  d_uniq_node_num_ptr,
                  sizeof(uint64_t),
                  cudaMemcpyDeviceToHost,
                  sample_stream_);
  cudaStreamSynchronize(sample_stream_);
  // 产生了足够多的node，采样结束
  VLOG(2) << "table capcity: " << train_table_cap_ << ", " << h_uniq_node_num
          << " used";
  if (h_uniq_node_num + len >= train_table_cap_) {
    return 1;
  }
  table_->insert(d_keys, len, d_uniq_node_num_ptr, sample_stream_);
  CUDA_CHECK(cudaStreamSynchronize(sample_stream_));
  return 0;
}

void GraphDataGenerator::DoWalk() {
  int device_id = place_.GetDeviceId();
  debug_gpu_memory_info(device_id, "DoWalk start");
  if (gpu_graph_training_) {
    FillWalkBuf();
  } else {
    FillInferBuf();
  }
  debug_gpu_memory_info(device_id, "DoWalk end");
}

void GraphDataGenerator::clear_gpu_mem() {
  d_len_per_row_.reset();
  d_sample_keys_.reset();
  d_prefix_sum_.reset();
  for (size_t i = 0; i < d_sampleidx2rows_.size(); i++) {
    d_sampleidx2rows_[i].reset();
  }
  delete table_;
}

int GraphDataGenerator::FillInferBuf() {
  platform::CUDADeviceGuard guard(gpuid_);
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  auto &global_infer_node_type_start =
      gpu_graph_ptr->global_infer_node_type_start_[gpuid_];
  auto &infer_cursor = gpu_graph_ptr->infer_cursor_[thread_id_];
  total_row_ = 0;
  if (infer_cursor < h_device_keys_len_.size()) {
    if (global_infer_node_type_start[infer_cursor] >=
        h_device_keys_len_[infer_cursor]) {
      infer_cursor++;
      if (infer_cursor >= h_device_keys_len_.size()) {
        return 0;
      }
    }
    size_t device_key_size = h_device_keys_len_[infer_cursor];
    total_row_ =
        (global_infer_node_type_start[infer_cursor] + infer_table_cap_ <=
         device_key_size)
            ? infer_table_cap_
            : device_key_size - global_infer_node_type_start[infer_cursor];

    host_vec_.resize(total_row_);
    uint64_t *d_type_keys =
        reinterpret_cast<uint64_t *>(d_device_keys_[infer_cursor]->ptr());
    cudaMemcpyAsync(host_vec_.data(),
                    d_type_keys + global_infer_node_type_start[infer_cursor],
                    sizeof(uint64_t) * total_row_,
                    cudaMemcpyDeviceToHost,
                    sample_stream_);
    cudaStreamSynchronize(sample_stream_);
    VLOG(1) << "cursor: " << infer_cursor
            << " start: " << global_infer_node_type_start[infer_cursor]
            << " num: " << total_row_;
    infer_node_start_ = global_infer_node_type_start[infer_cursor];
    global_infer_node_type_start[infer_cursor] += total_row_;
    infer_node_end_ = global_infer_node_type_start[infer_cursor];
    cursor_ = infer_cursor;
  }
  return 0;
}

void GraphDataGenerator::ClearSampleState() {
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  auto &finish_node_type = gpu_graph_ptr->finish_node_type_[gpuid_];
  auto &node_type_start = gpu_graph_ptr->node_type_start_[gpuid_];
  finish_node_type.clear();
  for (auto iter = node_type_start.begin(); iter != node_type_start.end();
       iter++) {
    iter->second = 0;
  }
}

int GraphDataGenerator::FillWalkBuf() {
=======
  int ret = gpu_graph_ptr->get_feature_of_nodes(
      gpuid_,
      (uint64_t *)d_walk->ptr(),     // NOLINT
      (uint64_t *)d_feature->ptr(),  // NOLINT
      buf_size_,
      slot_num_);
  return ret;
}

int GraphDataGenerator::FillWalkBuf(std::shared_ptr<phi::Allocation> d_walk) {
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  platform::CUDADeviceGuard guard(gpuid_);
  size_t once_max_sample_keynum = walk_degree_ * once_sample_startid_len_;
  ////////
  uint64_t *h_walk;
  uint64_t *h_sample_keys;
  int *h_offset2idx;
  int *h_len_per_row;
  uint64_t *h_prefix_sum;
  if (debug_mode_) {
    h_walk = new uint64_t[buf_size_];
    h_sample_keys = new uint64_t[once_max_sample_keynum];
    h_offset2idx = new int[once_max_sample_keynum];
    h_len_per_row = new int[once_max_sample_keynum];
    h_prefix_sum = new uint64_t[once_max_sample_keynum + 1];
  }
  ///////
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
<<<<<<< HEAD
  uint64_t *walk = reinterpret_cast<uint64_t *>(d_walk_->ptr());
  int *len_per_row = reinterpret_cast<int *>(d_len_per_row_->ptr());
  uint64_t *d_sample_keys = reinterpret_cast<uint64_t *>(d_sample_keys_->ptr());
  cudaMemsetAsync(walk, 0, buf_size_ * sizeof(uint64_t), sample_stream_);
  // cudaMemsetAsync(
  //     len_per_row, 0, once_max_sample_keynum * sizeof(int), sample_stream_);
  int sample_times = 0;
  int i = 0;
  total_row_ = 0;

  // 获取全局采样状态
  auto &first_node_type = gpu_graph_ptr->first_node_type_;
  auto &meta_path = gpu_graph_ptr->meta_path_;
  auto &node_type_start = gpu_graph_ptr->node_type_start_[gpuid_];
  auto &finish_node_type = gpu_graph_ptr->finish_node_type_[gpuid_];
  auto &type_to_index = gpu_graph_ptr->get_graph_type_to_index();
  auto &cursor = gpu_graph_ptr->cursor_[thread_id_];
  size_t node_type_len = first_node_type.size();
  int remain_size =
      buf_size_ - walk_degree_ * once_sample_startid_len_ * walk_len_;
  int total_samples = 0;

  while (i <= remain_size) {
    int cur_node_idx = cursor % node_type_len;
    int node_type = first_node_type[cur_node_idx];
    auto &path = meta_path[cur_node_idx];
    size_t start = node_type_start[node_type];
    VLOG(2) << "cur_node_idx = " << cur_node_idx
            << " meta_path.size = " << meta_path.size();
    // auto node_query_result = gpu_graph_ptr->query_node_list(
    //     gpuid_, node_type, start, once_sample_startid_len_);

    // int tmp_len = node_query_result.actual_sample_size;
    VLOG(2) << "choose start type: " << node_type;
    int type_index = type_to_index[node_type];
    size_t device_key_size = h_device_keys_len_[type_index];
=======
  uint64_t *walk = reinterpret_cast<uint64_t *>(d_walk->ptr());
  int *len_per_row = reinterpret_cast<int *>(d_len_per_row_->ptr());
  uint64_t *d_sample_keys = reinterpret_cast<uint64_t *>(d_sample_keys_->ptr());
  cudaMemsetAsync(walk, 0, buf_size_ * sizeof(uint64_t), stream_);
  cudaMemsetAsync(
      len_per_row, 0, once_max_sample_keynum * sizeof(int), stream_);
  int i = 0;
  int total_row = 0;
  size_t node_type_len = first_node_type_.size();
  int remain_size =
      buf_size_ - walk_degree_ * once_sample_startid_len_ * walk_len_;

  while (i <= remain_size) {
    int cur_node_idx = cursor_ % node_type_len;
    int node_type = first_node_type_[cur_node_idx];
    auto &path = meta_path_[cur_node_idx];
    size_t start = node_type_start_[node_type];
    // auto node_query_result = gpu_graph_ptr->query_node_list(
    //    gpuid_, node_type, start, once_sample_startid_len_);

    // int tmp_len = node_query_result.actual_sample_size;
    VLOG(2) << "choose start type: " << node_type;
    int type_index = type_to_index_[node_type];
    size_t device_key_size = h_device_keys_[type_index]->size();
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    VLOG(2) << "type: " << node_type << " size: " << device_key_size
            << " start: " << start;
    uint64_t *d_type_keys =
        reinterpret_cast<uint64_t *>(d_device_keys_[type_index]->ptr());
    int tmp_len = start + once_sample_startid_len_ > device_key_size
                      ? device_key_size - start
                      : once_sample_startid_len_;
<<<<<<< HEAD
    bool update = true;
    if (tmp_len == 0) {
      finish_node_type.insert(node_type);
      if (finish_node_type.size() == node_type_start.size()) {
        cursor = 0;
        epoch_finish_ = true;
        break;
      }
      cursor += 1;
      continue;
    }

    VLOG(2) << "gpuid = " << gpuid_ << " path[0] = " << path[0];
=======
    node_type_start_[node_type] = tmp_len + start;
    if (tmp_len == 0) {
      finish_node_type_.insert(node_type);
      if (finish_node_type_.size() == node_type_start_.size()) {
        break;
      }
      cursor_ += 1;
      continue;
    }
    // if (tmp_len == 0) {
    //  break;
    //}
    VLOG(2) << "i = " << i << " buf_size_ = " << buf_size_
            << " tmp_len = " << tmp_len << " cursor = " << cursor_
            << " once_max_sample_keynum = " << once_max_sample_keynum;
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    uint64_t *cur_walk = walk + i;

    NeighborSampleQuery q;
    q.initialize(gpuid_,
                 path[0],
                 (uint64_t)(d_type_keys + start),
                 walk_degree_,
                 tmp_len);
<<<<<<< HEAD
    auto sample_res = gpu_graph_ptr->graph_neighbor_sample_v3(q, false, true);
=======
    auto sample_res = gpu_graph_ptr->graph_neighbor_sample_v3(q, false);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f

    int step = 1;
    VLOG(2) << "sample edge type: " << path[0] << " step: " << 1;
    jump_rows_ = sample_res.total_sample_size;
<<<<<<< HEAD
    total_samples += sample_res.total_sample_size;
    VLOG(2) << "i = " << i << " start = " << start << " tmp_len = " << tmp_len
            << " cursor = " << node_type << " cur_node_idx = " << cur_node_idx
            << " jump row: " << jump_rows_;
    VLOG(2) << "jump_row: " << jump_rows_;
    if (jump_rows_ == 0) {
      node_type_start[node_type] = tmp_len + start;
      cursor += 1;
      continue;
    }

    if (FLAGS_gpugraph_storage_mode != GpuGraphStorageMode::WHOLE_HBM) {
      if (InsertTable(d_type_keys + start, tmp_len, d_uniq_node_num_) != 0) {
        VLOG(2) << "in step 0, insert key stage, table is full";
        update = false;
        break;
      }
      if (InsertTable(sample_res.actual_val,
                      sample_res.total_sample_size,
                      d_uniq_node_num_) != 0) {
        VLOG(2) << "in step 0, insert sample res stage, table is full";
        update = false;
        break;
      }
    }
=======
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    FillOneStep(d_type_keys + start,
                cur_walk,
                tmp_len,
                sample_res,
                walk_degree_,
                step,
                len_per_row);
<<<<<<< HEAD
=======
    VLOG(2) << "jump_row: " << jump_rows_;
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    /////////
    if (debug_mode_) {
      cudaMemcpy(
          h_walk, walk, buf_size_ * sizeof(uint64_t), cudaMemcpyDeviceToHost);
      for (int xx = 0; xx < buf_size_; xx++) {
        VLOG(2) << "h_walk[" << xx << "]: " << h_walk[xx];
      }
    }
<<<<<<< HEAD

    VLOG(2) << "sample, step=" << step << " sample_keys=" << tmp_len
            << " sample_res_len=" << sample_res.total_sample_size;

=======
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    /////////
    step++;
    size_t path_len = path.size();
    for (; step < walk_len_; step++) {
      if (sample_res.total_sample_size == 0) {
<<<<<<< HEAD
        VLOG(2) << "sample finish, step=" << step;
=======
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
        break;
      }
      auto sample_key_mem = sample_res.actual_val_mem;
      uint64_t *sample_keys_ptr =
          reinterpret_cast<uint64_t *>(sample_key_mem->ptr());
      int edge_type_id = path[(step - 1) % path_len];
      VLOG(2) << "sample edge type: " << edge_type_id << " step: " << step;
      q.initialize(gpuid_,
                   edge_type_id,
                   (uint64_t)sample_keys_ptr,
                   1,
                   sample_res.total_sample_size);
<<<<<<< HEAD
      int sample_key_len = sample_res.total_sample_size;
      sample_res = gpu_graph_ptr->graph_neighbor_sample_v3(q, false, true);
      total_samples += sample_res.total_sample_size;
      if (FLAGS_gpugraph_storage_mode != GpuGraphStorageMode::WHOLE_HBM) {
        if (InsertTable(sample_res.actual_val,
                        sample_res.total_sample_size,
                        d_uniq_node_num_) != 0) {
          VLOG(2) << "in step: " << step << ", table is full";
          update = false;
          break;
        }
      }
      FillOneStep(d_type_keys + start,
                  cur_walk,
                  sample_key_len,
=======
      sample_res = gpu_graph_ptr->graph_neighbor_sample_v3(q, false);

      FillOneStep(d_type_keys + start,
                  cur_walk,
                  sample_res.total_sample_size,
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
                  sample_res,
                  1,
                  step,
                  len_per_row);
      if (debug_mode_) {
        cudaMemcpy(
            h_walk, walk, buf_size_ * sizeof(uint64_t), cudaMemcpyDeviceToHost);
        for (int xx = 0; xx < buf_size_; xx++) {
          VLOG(2) << "h_walk[" << xx << "]: " << h_walk[xx];
        }
      }
<<<<<<< HEAD

      VLOG(2) << "sample, step=" << step << " sample_keys=" << sample_key_len
              << " sample_res_len=" << sample_res.total_sample_size;
    }
    // 此时更新全局采样状态
    if (update == true) {
      node_type_start[node_type] = tmp_len + start;
      i += jump_rows_ * walk_len_;
      total_row_ += jump_rows_;
      cursor += 1;
      sample_times++;
    } else {
      VLOG(2) << "table is full, not update stat!";
      break;
    }
  }
  buf_state_.Reset(total_row_);
  int *d_random_row = reinterpret_cast<int *>(d_random_row_->ptr());

  thrust::random::default_random_engine engine(shuffle_seed_);
  const auto &exec_policy = thrust::cuda::par.on(sample_stream_);
  thrust::counting_iterator<int> cnt_iter(0);
  thrust::shuffle_copy(exec_policy,
                       cnt_iter,
                       cnt_iter + total_row_,
                       thrust::device_pointer_cast(d_random_row),
                       engine);

  cudaStreamSynchronize(sample_stream_);
  shuffle_seed_ = engine();

  if (debug_mode_) {
    int *h_random_row = new int[total_row_ + 10];
    cudaMemcpy(h_random_row,
               d_random_row,
               total_row_ * sizeof(int),
               cudaMemcpyDeviceToHost);
    for (int xx = 0; xx < total_row_; xx++) {
=======
    }
    // cursor_ += tmp_len;
    i += jump_rows_ * walk_len_;
    total_row += jump_rows_;
    cursor_ += 1;
  }
  buf_state_.Reset(total_row);
  int *d_random_row = reinterpret_cast<int *>(d_random_row_->ptr());

  thrust::random::default_random_engine engine(shuffle_seed_);
  const auto &exec_policy = thrust::cuda::par.on(stream_);
  thrust::counting_iterator<int> cnt_iter(0);
  thrust::shuffle_copy(exec_policy,
                       cnt_iter,
                       cnt_iter + total_row,
                       thrust::device_pointer_cast(d_random_row),
                       engine);

  cudaStreamSynchronize(stream_);
  shuffle_seed_ = engine();

  if (debug_mode_) {
    int *h_random_row = new int[total_row + 10];
    cudaMemcpy(h_random_row,
               d_random_row,
               total_row * sizeof(int),
               cudaMemcpyDeviceToHost);
    for (int xx = 0; xx < total_row; xx++) {
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
      VLOG(2) << "h_random_row[" << xx << "]: " << h_random_row[xx];
    }
    delete[] h_random_row;
    delete[] h_walk;
    delete[] h_sample_keys;
    delete[] h_offset2idx;
    delete[] h_len_per_row;
    delete[] h_prefix_sum;
  }
<<<<<<< HEAD
  if (FLAGS_gpugraph_storage_mode != GpuGraphStorageMode::WHOLE_HBM) {
    // table_->prefetch(cudaCpuDeviceId, sample_stream_);
    // thrust::pair<uint64_t, uint64_t> *kv = table_->data();
    // size_t size = table_->size();
    // uint64_t unused_key = std::numeric_limits<uint64_t>::max();
    // for (size_t i = 0; i < size; i++) {
    //   if (kv[i].first == unused_key) {
    //     continue;
    //   }
    //   host_vec_.push_back(kv[i].first);
    // }

    uint64_t h_uniq_node_num = 0;
    uint64_t *d_uniq_node_num =
        reinterpret_cast<uint64_t *>(d_uniq_node_num_->ptr());
    cudaMemcpyAsync(&h_uniq_node_num,
                    d_uniq_node_num,
                    sizeof(uint64_t),
                    cudaMemcpyDeviceToHost,
                    sample_stream_);
    cudaStreamSynchronize(sample_stream_);
    VLOG(2) << "h_uniq_node_num: " << h_uniq_node_num;
    // 临时显存, 存储去重后的nodeid
    auto d_uniq_node = memory::AllocShared(
        place_,
        h_uniq_node_num * sizeof(uint64_t),
        phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
    uint64_t *d_uniq_node_ptr =
        reinterpret_cast<uint64_t *>(d_uniq_node->ptr());

    auto d_node_cursor = memory::AllocShared(
        place_,
        sizeof(uint64_t),
        phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));

    uint64_t *d_node_cursor_ptr =
        reinterpret_cast<uint64_t *>(d_node_cursor->ptr());
    cudaMemsetAsync(d_node_cursor_ptr, 0, sizeof(uint64_t), sample_stream_);
    // uint64_t unused_key = std::numeric_limits<uint64_t>::max();
    table_->get_keys(d_uniq_node_ptr, d_node_cursor_ptr, sample_stream_);

    cudaStreamSynchronize(sample_stream_);

    host_vec_.resize(h_uniq_node_num);
    cudaMemcpyAsync(host_vec_.data(),
                    d_uniq_node_ptr,
                    sizeof(uint64_t) * h_uniq_node_num,
                    cudaMemcpyDeviceToHost,
                    sample_stream_);
    cudaStreamSynchronize(sample_stream_);

    VLOG(0) << "sample_times:" << sample_times << ", d_walk_size:" << buf_size_
            << ", d_walk_offset:" << i << ", total_rows:" << total_row_
            << ", total_samples:" << total_samples
            << ", h_uniq_node_num:" << h_uniq_node_num;
  }
  return total_row_ != 0;
}

void GraphDataGenerator::SetFeedVec(std::vector<LoDTensor *> feed_vec) {
  feed_vec_ = feed_vec;
}
void GraphDataGenerator::AllocResource(int thread_id,
                                       std::vector<LoDTensor *> feed_vec) {
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  gpuid_ = gpu_graph_ptr->device_id_mapping[thread_id];
  thread_id_ = thread_id;
  place_ = platform::CUDAPlace(gpuid_);
  debug_gpu_memory_info(gpuid_, "AllocResource start");

  platform::CUDADeviceGuard guard(gpuid_);
  if (FLAGS_gpugraph_storage_mode != GpuGraphStorageMode::WHOLE_HBM) {
    table_ = new HashTable<uint64_t, uint64_t>(
        train_table_cap_ / FLAGS_gpugraph_hbm_table_load_factor);
  }
  VLOG(1) << "AllocResource gpuid " << gpuid_
          << " feed_vec.size: " << feed_vec.size()
          << " table cap: " << train_table_cap_;
  sample_stream_ = gpu_graph_ptr->get_local_stream(gpuid_);
  train_stream_ = dynamic_cast<phi::GPUContext *>(
                      platform::DeviceContextPool::Instance().Get(place_))
                      ->stream();
  // feed_vec_ = feed_vec;
  slot_num_ = (feed_vec.size() - 3) / 2;

  // infer_node_type_start_ = std::vector<int>(h_device_keys_.size(), 0);
  // for (size_t i = 0; i < h_device_keys_.size(); i++) {
  //   for (size_t j = 0; j < h_device_keys_[i]->size(); j++) {
  //     VLOG(3) << "h_device_keys_[" << i << "][" << j
  //             << "] = " << (*(h_device_keys_[i]))[j];
  //   }
  //   auto buf = memory::AllocShared(
  //       place_, h_device_keys_[i]->size() * sizeof(uint64_t));
  //   d_device_keys_.push_back(buf);
  //   CUDA_CHECK(cudaMemcpyAsync(buf->ptr(),
  //                              h_device_keys_[i]->data(),
  //                              h_device_keys_[i]->size() * sizeof(uint64_t),
  //                              cudaMemcpyHostToDevice,
  //                              stream_));
  // }
  auto &d_graph_all_type_keys = gpu_graph_ptr->d_graph_all_type_total_keys_;
  auto &h_graph_all_type_keys_len = gpu_graph_ptr->h_graph_all_type_keys_len_;

  for (size_t i = 0; i < d_graph_all_type_keys.size(); i++) {
    d_device_keys_.push_back(d_graph_all_type_keys[i][thread_id]);
    h_device_keys_len_.push_back(h_graph_all_type_keys_len[i][thread_id]);
  }
  VLOG(2) << "h_device_keys size: " << h_device_keys_len_.size();

  size_t once_max_sample_keynum = walk_degree_ * once_sample_startid_len_;
  d_prefix_sum_ = memory::AllocShared(
      place_,
      (once_max_sample_keynum + 1) * sizeof(int),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
  int *d_prefix_sum_ptr = reinterpret_cast<int *>(d_prefix_sum_->ptr());
  cudaMemsetAsync(d_prefix_sum_ptr,
                  0,
                  (once_max_sample_keynum + 1) * sizeof(int),
                  sample_stream_);
  cursor_ = 0;
  jump_rows_ = 0;
  d_uniq_node_num_ = memory::AllocShared(
      place_,
      sizeof(uint64_t),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
  cudaMemsetAsync(d_uniq_node_num_->ptr(), 0, sizeof(uint64_t), sample_stream_);

  d_walk_ = memory::AllocShared(
      place_,
      buf_size_ * sizeof(uint64_t),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
  cudaMemsetAsync(
      d_walk_->ptr(), 0, buf_size_ * sizeof(uint64_t), sample_stream_);
  d_sample_keys_ = memory::AllocShared(
      place_,
      once_max_sample_keynum * sizeof(uint64_t),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));

  d_sampleidx2rows_.push_back(memory::AllocShared(
      place_,
      once_max_sample_keynum * sizeof(int),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_))));
  d_sampleidx2rows_.push_back(memory::AllocShared(
      place_,
      once_max_sample_keynum * sizeof(int),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_))));
  cur_sampleidx2row_ = 0;

  d_len_per_row_ = memory::AllocShared(
      place_,
      once_max_sample_keynum * sizeof(int),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
=======
  return total_row != 0;
}

void GraphDataGenerator::AllocResource(
    const paddle::platform::Place &place,
    std::vector<phi::DenseTensor *> feed_vec) {
  place_ = place;
  gpuid_ = place_.GetDeviceId();
  VLOG(3) << "gpuid " << gpuid_;
  stream_ = dynamic_cast<phi::GPUContext *>(
                platform::DeviceContextPool::Instance().Get(place))
                ->stream();
  feed_vec_ = feed_vec;
  slot_num_ = (feed_vec_.size() - 3) / 2;

  // d_device_keys_.resize(h_device_keys_.size());
  VLOG(2) << "h_device_keys size: " << h_device_keys_.size();
  infer_node_type_start_ = std::vector<int>(h_device_keys_.size(), 0);
  for (size_t i = 0; i < h_device_keys_.size(); i++) {
    for (size_t j = 0; j < h_device_keys_[i]->size(); j++) {
      VLOG(3) << "h_device_keys_[" << i << "][" << j
              << "] = " << (*(h_device_keys_[i]))[j];
    }
    auto buf = memory::AllocShared(
        place_, h_device_keys_[i]->size() * sizeof(uint64_t));
    d_device_keys_.push_back(buf);
    CUDA_CHECK(cudaMemcpyAsync(buf->ptr(),
                               h_device_keys_[i]->data(),
                               h_device_keys_[i]->size() * sizeof(uint64_t),
                               cudaMemcpyHostToDevice,
                               stream_));
  }
  // h_device_keys_ = h_device_keys;
  // device_key_size_ = h_device_keys_->size();
  // d_device_keys_ =
  //    memory::AllocShared(place_, device_key_size_ * sizeof(int64_t));
  // CUDA_CHECK(cudaMemcpyAsync(d_device_keys_->ptr(), h_device_keys_->data(),
  //                           device_key_size_ * sizeof(int64_t),
  //                           cudaMemcpyHostToDevice, stream_));
  size_t once_max_sample_keynum = walk_degree_ * once_sample_startid_len_;
  d_prefix_sum_ =
      memory::AllocShared(place_, (once_max_sample_keynum + 1) * sizeof(int));
  int *d_prefix_sum_ptr = reinterpret_cast<int *>(d_prefix_sum_->ptr());
  cudaMemsetAsync(
      d_prefix_sum_ptr, 0, (once_max_sample_keynum + 1) * sizeof(int), stream_);
  cursor_ = 0;
  jump_rows_ = 0;
  d_walk_ = memory::AllocShared(place_, buf_size_ * sizeof(uint64_t));
  cudaMemsetAsync(d_walk_->ptr(), 0, buf_size_ * sizeof(uint64_t), stream_);
  if (!FLAGS_enable_opt_get_features && slot_num_ > 0) {
    d_feature_ =
        memory::AllocShared(place_, buf_size_ * slot_num_ * sizeof(uint64_t));
    cudaMemsetAsync(
        d_feature_->ptr(), 0, buf_size_ * sizeof(uint64_t), stream_);
  }
  d_sample_keys_ =
      memory::AllocShared(place_, once_max_sample_keynum * sizeof(uint64_t));

  d_sampleidx2rows_.push_back(
      memory::AllocShared(place_, once_max_sample_keynum * sizeof(int)));
  d_sampleidx2rows_.push_back(
      memory::AllocShared(place_, once_max_sample_keynum * sizeof(int)));
  cur_sampleidx2row_ = 0;

  d_len_per_row_ =
      memory::AllocShared(place_, once_max_sample_keynum * sizeof(int));
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  for (int i = -window_; i < 0; i++) {
    window_step_.push_back(i);
  }
  for (int i = 0; i < window_; i++) {
    window_step_.push_back(i + 1);
  }
  buf_state_.Init(batch_size_, walk_len_, &window_step_);
  d_random_row_ = memory::AllocShared(
      place_,
<<<<<<< HEAD
      (once_sample_startid_len_ * walk_degree_ * repeat_time_) * sizeof(int),
      phi::Stream(reinterpret_cast<phi::StreamId>(sample_stream_)));
=======
      (once_sample_startid_len_ * walk_degree_ * repeat_time_) * sizeof(int));
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  shuffle_seed_ = 0;

  ins_buf_pair_len_ = 0;
  d_ins_buf_ =
      memory::AllocShared(place_, (batch_size_ * 2 * 2) * sizeof(uint64_t));
<<<<<<< HEAD
  d_pair_num_ = memory::AllocShared(place_, sizeof(int));

  d_slot_tensor_ptr_ =
      memory::AllocShared(place_, slot_num_ * sizeof(uint64_t *));
  d_slot_lod_tensor_ptr_ =
      memory::AllocShared(place_, slot_num_ * sizeof(uint64_t *));

  cudaStreamSynchronize(sample_stream_);

  debug_gpu_memory_info(gpuid_, "AllocResource end");
}

void GraphDataGenerator::AllocTrainResource(int thread_id) {
  if (slot_num_ > 0) {
    platform::CUDADeviceGuard guard(gpuid_);
    d_feature_size_list_buf_ =
        memory::AllocShared(place_, (batch_size_ * 2) * sizeof(uint32_t));
    d_feature_size_prefixsum_buf_ =
        memory::AllocShared(place_, (batch_size_ * 2 + 1) * sizeof(uint32_t));
  }
=======
  if (slot_num_ > 0) {
    d_feature_buf_ = memory::AllocShared(
        place_, (batch_size_ * 2 * 2) * slot_num_ * sizeof(uint64_t));
  }
  d_pair_num_ = memory::AllocShared(place_, sizeof(int));
  if (FLAGS_enable_opt_get_features && slot_num_ > 0) {
    d_slot_tensor_ptr_ =
        memory::AllocShared(place_, slot_num_ * sizeof(uint64_t *));
    d_slot_lod_tensor_ptr_ =
        memory::AllocShared(place_, slot_num_ * sizeof(uint64_t *));
  }

  cudaStreamSynchronize(stream_);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
}

void GraphDataGenerator::SetConfig(
    const paddle::framework::DataFeedDesc &data_feed_desc) {
  auto graph_config = data_feed_desc.graph_config();
  walk_degree_ = graph_config.walk_degree();
  walk_len_ = graph_config.walk_len();
  window_ = graph_config.window();
  once_sample_startid_len_ = graph_config.once_sample_startid_len();
  debug_mode_ = graph_config.debug_mode();
  gpu_graph_training_ = graph_config.gpu_graph_training();
  if (debug_mode_ || !gpu_graph_training_) {
    batch_size_ = graph_config.batch_size();
  } else {
    batch_size_ = once_sample_startid_len_;
  }
  repeat_time_ = graph_config.sample_times_one_chunk();
  buf_size_ =
      once_sample_startid_len_ * walk_len_ * walk_degree_ * repeat_time_;
<<<<<<< HEAD
  train_table_cap_ = graph_config.train_table_cap();
  infer_table_cap_ = graph_config.infer_table_cap();
  epoch_finish_ = false;
  VLOG(1) << "Confirm GraphConfig, walk_degree : " << walk_degree_
          << ", walk_len : " << walk_len_ << ", window : " << window_
          << ", once_sample_startid_len : " << once_sample_startid_len_
          << ", sample_times_one_chunk : " << repeat_time_
          << ", batch_size: " << batch_size_
          << ", train_table_cap: " << train_table_cap_
          << ", infer_table_cap: " << infer_table_cap_;
  std::string first_node_type = graph_config.first_node_type();
  std::string meta_path = graph_config.meta_path();
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  debug_gpu_memory_info("init_conf start");
  gpu_graph_ptr->init_conf(first_node_type, meta_path);
  debug_gpu_memory_info("init_conf end");
};
=======
  VLOG(2) << "Confirm GraphConfig, walk_degree : " << walk_degree_
          << ", walk_len : " << walk_len_ << ", window : " << window_
          << ", once_sample_startid_len : " << once_sample_startid_len_
          << ", sample_times_one_chunk : " << repeat_time_
          << ", batch_size: " << batch_size_;
  std::string first_node_type = graph_config.first_node_type();
  std::string meta_path = graph_config.meta_path();
  auto gpu_graph_ptr = GraphGpuWrapper::GetInstance();
  auto edge_to_id = gpu_graph_ptr->edge_to_id;
  auto node_to_id = gpu_graph_ptr->feature_to_id;
  // parse first_node_type
  auto node_types =
      paddle::string::split_string<std::string>(first_node_type, ";");
  VLOG(2) << "node_types: " << first_node_type;
  finish_node_type_.clear();
  node_type_start_.clear();
  for (auto &type : node_types) {
    auto iter = node_to_id.find(type);
    PADDLE_ENFORCE_NE(
        iter,
        node_to_id.end(),
        platform::errors::NotFound("(%s) is not found in node_to_id.", type));
    VLOG(2) << "node_to_id[" << type << "] = " << iter->second;
    first_node_type_.push_back(iter->second);
    node_type_start_[iter->second] = 0;
  }
  meta_path_.resize(first_node_type_.size());
  auto meta_paths = paddle::string::split_string<std::string>(meta_path, ";");

  for (size_t i = 0; i < meta_paths.size(); i++) {
    auto path = meta_paths[i];
    auto nodes = paddle::string::split_string<std::string>(path, "-");
    for (auto &node : nodes) {
      auto iter = edge_to_id.find(node);
      PADDLE_ENFORCE_NE(
          iter,
          edge_to_id.end(),
          platform::errors::NotFound("(%s) is not found in edge_to_id.", node));
      VLOG(2) << "edge_to_id[" << node << "] = " << iter->second;
      meta_path_[i].push_back(iter->second);
    }
  }
}
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f

}  // namespace framework
}  // namespace paddle
#endif
