/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "gtest/gtest.h"

#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/framework/op_info.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/scope.h"
#include "paddle/fluid/operators/elementwise/elementwise_op_function.h"
#include "paddle/fluid/operators/math/math_function.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/fluid/platform/init.h"

namespace paddle {
namespace framework {

template <typename T>
struct AddFunctor {
  inline HOSTDEVICE T operator()(T a, T b) const { return a + b; }
};

class OpKernelTestProtoAndCheckerMaker : public OpProtoAndCheckerMaker {
 public:
  void Make() {
    AddInput("input", "input1 of test op");
    AddOutput("output", "output of test op");
    AddAttr<bool>("use_gpu", "force to use gpu kernel").SetDefault(false);
    AddComment("This is test op");
  }
};

class TestOpWithKernel : public OperatorWithKernel {
 public:
  using OperatorWithKernel::OperatorWithKernel;

 protected:
  void InferShape(framework::InferShapeContext* ctx) const override {}
  OpKernelType GetExpectedKernelType(
      const ExecutionContext& ctx) const override {
    if (Attr<bool>("use_gpu")) {
      VLOG(3) << "force use gpu kernel";
      return OpKernelType(proto::VarType::FP32, platform::CUDAPlace(0));
    } else {
      VLOG(3) << "use default kernel";
      return OpKernelType(proto::VarType::FP32,
                          ctx.Input<Tensor>("input")->place());
    }
  }
};

template <typename DeviceContext, typename T>
class TestKernel : public OpKernel<float> {
 public:
  void Compute(const ExecutionContext& ctx) const {
    std::cout << ctx.op().DebugString() << std::endl;

    const Tensor* input = ctx.Input<Tensor>("input");

    std::cout << "input place:" << input->place() << std::endl;
    auto* output = ctx.Output<framework::LoDTensor>("output");
    output->Resize(input->dims());
    output->mutable_data<T>(ctx.GetPlace());

    operators::TransformFunctor<AddFunctor<T>, T, DeviceContext> functor(
        input, input, output, ctx.template device_context<DeviceContext>(),
        AddFunctor<T>());
    functor.Run();
  }
};

}  // namespace framework
}  // namespace paddle

REGISTER_OP_WITHOUT_GRADIENT(
    test_op, paddle::framework::TestOpWithKernel,
    paddle::framework::OpKernelTestProtoAndCheckerMaker);
REGISTER_OP_CPU_KERNEL(
    test_op,
    paddle::framework::TestKernel<paddle::platform::CPUDeviceContext, float>);
REGISTER_OP_CUDA_KERNEL(
    test_op,
    paddle::framework::TestKernel<paddle::platform::CUDADeviceContext, float>);

static void BuildVar(const std::string& param_name,
                     std::initializer_list<const char*> arguments,
                     paddle::framework::proto::OpDesc::Var* var) {
  var->set_parameter(param_name);
  for (auto& arg_name : arguments) {
    *var->mutable_arguments()->Add() = arg_name;
  }
}

TEST(Operator, CPUtoGPU) {
  paddle::framework::InitDevices(true);

  paddle::framework::Scope scope;
  paddle::platform::CPUPlace cpu_place;

  // create an op to run on CPU
  paddle::framework::proto::OpDesc cpu_op_desc;
  cpu_op_desc.set_type("test_op");
  BuildVar("input", {"IN1"}, cpu_op_desc.add_inputs());
  BuildVar("output", {"OUT1"}, cpu_op_desc.add_outputs());

  auto cpu_op = paddle::framework::OpRegistry::CreateOp(cpu_op_desc);
  // prepare input
  auto* in_t = scope.Var("IN1")->GetMutable<paddle::framework::LoDTensor>();
  auto* src_ptr =
      in_t->mutable_data<float>({2, 3}, paddle::platform::CPUPlace());
  for (int i = 0; i < 2 * 3; ++i) {
    src_ptr[i] = static_cast<float>(i);
  }

  // get output
  auto* output = scope.Var("OUT1");
  cpu_op->Run(scope, cpu_place);

  auto* output_ptr = output->Get<paddle::framework::LoDTensor>().data<float>();
  for (int i = 0; i < 2 * 3; ++i) {
    ASSERT_EQ(output_ptr[i], static_cast<float>(i) * 2);
  }

  // create an op to run on GPU
  paddle::framework::proto::OpDesc gpu_op_desc;
  gpu_op_desc.set_type("test_op");
  BuildVar("input", {"OUT1"}, gpu_op_desc.add_inputs());
  BuildVar("output", {"OUT2"}, gpu_op_desc.add_outputs());

  auto attr = gpu_op_desc.mutable_attrs()->Add();
  attr->set_name("use_gpu");
  attr->set_type(paddle::framework::proto::AttrType::BOOLEAN);
  attr->set_b(true);

  auto gpu_op = paddle::framework::OpRegistry::CreateOp(gpu_op_desc);

  paddle::platform::CUDAPlace cuda_place(0);
  // get output
  auto* output2 = scope.Var("OUT2");
  gpu_op->Run(scope, cuda_place);
  VLOG(3) << "after gpu_op run";

  // auto* output2_ptr = output2->Get<LoDTensor>().data<float>();
  paddle::platform::DeviceContextPool& pool =
      paddle::platform::DeviceContextPool::Instance();
  auto dev_ctx = pool.Get(cuda_place);

  paddle::framework::Tensor output_tensor;
  paddle::framework::TensorCopy(output2->Get<paddle::framework::LoDTensor>(),
                                paddle::platform::CPUPlace(), *dev_ctx,
                                &output_tensor);

  dev_ctx->Wait();
  float* output2_ptr = output_tensor.data<float>();
  for (int i = 0; i < 2 * 3; ++i) {
    ASSERT_EQ(output2_ptr[i], static_cast<float>(i) * 4);
  }
}
