
#include <hip/hip_runtime.h>
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_PSLIB
#include <algorithm>
#include <ctime>
#include <memory>
#include <numeric>
#include "paddle/fluid/framework/fleet/ps_gpu_wrapper.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace framework {

__global__ void PullCopy(float** dest, const FeatureValue* src,
                         const int64_t* len, int hidden, int slot_num,
                         int total_len, uint64_t** keys) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    if (*(keys[x] + y) == 0) {
      *(dest[x] + y * hidden) = 0;
      *(dest[x] + y * hidden + 1) = 0;
      *(dest[x] + y * hidden + 2) = 0;
    } else {
      *(dest[x] + y * hidden) = (src + i)->show;
      *(dest[x] + y * hidden + 1) = (src + i)->clk;
      *(dest[x] + y * hidden + 2) = (src + i)->lr;
    }
    if ((src + i)->mf_size == 0 || *(keys[x] + y) == 0) {
      for (int j = 0; j < 8; j++) {
        *(dest[x] + y * hidden + 3 + j) = 0;
      }
    } else {
      for (int j = 0; j < 8; j++) {
        *(dest[x] + y * hidden + 3 + j) = (src + i)->mf[1 + j];
      }
    }
  }
}

__global__ void CopyKeysKernel(uint64_t** src_keys, uint64_t* dest_total_keys,
                               const int64_t* len, int slot_num,
                               int total_len) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    dest_total_keys[i] = src_keys[x][y];
  }
}

__global__ void PushCopy(FeaturePushValue* dest, float** src,
                         int64_t* len, int hidden, int slot_num, int total_len,
                         int bs, int* slot_vector) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[low - 1] : 0);
    (dest + i)->slot = slot_vector[x];
    (dest + i)->show = *(src[x] + y * hidden);
    (dest + i)->clk = *(src[x] + y * hidden + 1);
    (dest + i)->lr_g = *(src[x] + y * hidden + 2) * -1. * bs;
    for (int j = 0; j < 8; j++) {
      (dest + i)->mf_g[j] = *(src[x] + y * hidden + 3 + j) * -1. * bs;
    }
  }
}

void PSGPUWrapper::CopyForPull(const paddle::platform::Place& place,
                             uint64_t** gpu_keys,
                             const std::vector<float*>& values,
                             const FeatureValue* total_values_gpu,
                             const int64_t* gpu_len, const int slot_num,
                             const int hidden_size,
                             const int64_t total_length) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        boost::get<platform::CUDAPlace>(place)))
                    ->stream();
  auto buf_value = memory::AllocShared(place, values.size() * sizeof(float*));
  float** gpu_values = reinterpret_cast<float**>(buf_value->ptr());
  cudaMemcpy(gpu_values, values.data(), values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);

  PullCopy<<<(total_length + 512 - 1) / 512, 512, 0, stream>>>(
      gpu_values, total_values_gpu, gpu_len, hidden_size, slot_num,
      total_length, gpu_keys);
  cudaStreamSynchronize(stream);
}

void PSGPUWrapper::CopyKeys(const paddle::platform::Place& place,
                          uint64_t** origin_keys, uint64_t* total_keys,
                          const int64_t* gpu_len, int slot_num, int total_len) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        boost::get<platform::CUDAPlace>(place)))
                    ->stream();
  CopyKeysKernel<<<(total_len + 512 - 1) / 512, 512, 0, stream>>>(
      origin_keys, total_keys, gpu_len, slot_num, total_len);
  cudaStreamSynchronize(stream);
}

void PSGPUWrapper::CopyForPush(const paddle::platform::Place& place,
                             const std::vector<const float*>& grad_values,
                             FeaturePushValue* total_grad_values_gpu,
                             const std::vector<int64_t>& slot_lengths,
                             const int hidden_size, const int64_t total_length,
                             const int batch_size) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        boost::get<platform::CUDAPlace>(place)))
                    ->stream();
  auto slot_lengths_lod = slot_lengths;
  for (int i = 1; i < slot_lengths_lod.size(); i++) {
    slot_lengths_lod[i] += slot_lengths_lod[i - 1];
  }
  auto buf_grad_value =
      memory::AllocShared(place, grad_values.size() * sizeof(float*));
  auto buf_length =
      memory::AllocShared(place, slot_lengths.size() * sizeof(int64_t));
  auto buf_slot_vector =
      memory::AllocShared(place, slot_lengths_lod.size() * sizeof(int));

  float** gpu_values = reinterpret_cast<float**>(buf_grad_value->ptr());
  int64_t* gpu_len = reinterpret_cast<int64_t*>(buf_length->ptr());
  int* d_slot_vector = reinterpret_cast<int*>(buf_slot_vector->ptr());

  cudaMemcpy(gpu_values, grad_values.data(),
             grad_values.size() * sizeof(float*), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_len, slot_lengths_lod.data(),
             slot_lengths.size() * sizeof(int64_t), cudaMemcpyHostToDevice);
  cudaMemcpy(d_slot_vector, slot_vector_.data(),
             slot_lengths_lod.size() * sizeof(int), cudaMemcpyHostToDevice);

  PushCopy<<<(total_length + 512 - 1) / 512, 512, 0, stream>>>(
      total_grad_values_gpu, gpu_values, gpu_len, hidden_size,
      slot_lengths.size(), total_length, batch_size, d_slot_vector);
  cudaStreamSynchronize(stream);
}
}  // end namespace framework
}  // end namespace paddle
#endif
