
#include <hip/hip_runtime.h>
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_BOX_PS
#include <algorithm>
#include <ctime>
#include <memory>
#include <numeric>
#include "paddle/fluid/framework/fleet/box_wrapper.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace framework {
#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

__global__ void PullCopy(float** dest, const boxps::FeatureValueGpu* src,
                         const int64_t* len, int hidden, int slot_num,
                         int total_len, uint64_t** keys) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    if (*(keys[x] + y) == 0) {
      *(dest[x] + y * hidden) = 0;
      *(dest[x] + y * hidden + 1) = 0;
      *(dest[x] + y * hidden + 2) = 0;
    } else {
      *(dest[x] + y * hidden) = (src + i)->show;
      *(dest[x] + y * hidden + 1) = (src + i)->clk;
      *(dest[x] + y * hidden + 2) = (src + i)->embed_w;
    }
    if ((src + i)->embedding_size == 0 || *(keys[x] + y) == 0) {
      for (int j = 0; j < 8; j++) {
        *(dest[x] + y * hidden + 3 + j) = 0;
      }
    } else {
      for (int j = 0; j < 8; j++) {
        *(dest[x] + y * hidden + 3 + j) = (src + i)->embedx[1 + j];
      }
    }
  }
}

__global__ void CopyKeysKernel(uint64_t** src_keys, uint64_t* dest_total_keys,
                               const int64_t* len, int slot_num,
                               int total_len) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    dest_total_keys[i] = src_keys[x][y];
  }
}

__global__ void PushCopy(boxps::FeaturePushValueGpu* dest, float** src,
                         int64_t* len, int hidden, int slot_num, int total_len,
                         int bs, int* slot_vector) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[low - 1] : 0);
    (dest + i)->slot = slot_vector[x];
    (dest + i)->show = *(src[x] + y * hidden);
    (dest + i)->clk = *(src[x] + y * hidden + 1);
    (dest + i)->embed_g = *(src[x] + y * hidden + 2) * -1. * bs;
    for (int j = 0; j < 8; j++) {
      (dest + i)->embedx_g[j] = *(src[x] + y * hidden + 3 + j) * -1. * bs;
    }
  }
}

void BoxWrapper::CopyForPull(const paddle::platform::Place& place,
                             uint64_t** gpu_keys,
                             const std::vector<float*>& values,
                             const boxps::FeatureValueGpu* total_values_gpu,
                             const int64_t* gpu_len, const int slot_num,
                             const int hidden_size,
                             const int64_t total_length) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  auto buf_value = memory::AllocShared(place, values.size() * sizeof(float*));
  float** gpu_values = reinterpret_cast<float**>(buf_value->ptr());
  cudaMemcpy(gpu_values, values.data(), values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);

  PullCopy<<<(total_length + 512 - 1) / 512, 512, 0, stream>>>(
      gpu_values, total_values_gpu, gpu_len, hidden_size, slot_num,
      total_length, gpu_keys);
  cudaStreamSynchronize(stream);
}

void BoxWrapper::CopyKeys(const paddle::platform::Place& place,
                          uint64_t** origin_keys, uint64_t* total_keys,
                          const int64_t* gpu_len, int slot_num, int total_len) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  CopyKeysKernel<<<(total_len + 512 - 1) / 512, 512, 0, stream>>>(
      origin_keys, total_keys, gpu_len, slot_num, total_len);
  cudaStreamSynchronize(stream);
}

void BoxWrapper::CopyForPush(const paddle::platform::Place& place,
                             const std::vector<const float*>& grad_values,
                             boxps::FeaturePushValueGpu* total_grad_values_gpu,
                             const std::vector<int64_t>& slot_lengths,
                             const int hidden_size, const int64_t total_length,
                             const int batch_size) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  auto slot_lengths_lod = slot_lengths;
  for (int i = 1; i < slot_lengths_lod.size(); i++) {
    slot_lengths_lod[i] += slot_lengths_lod[i - 1];
  }
  auto buf_grad_value =
      memory::AllocShared(place, grad_values.size() * sizeof(float*));
  auto buf_length =
      memory::AllocShared(place, slot_lengths.size() * sizeof(int64_t));
  auto buf_slot_vector =
      memory::AllocShared(place, slot_lengths_lod.size() * sizeof(int));

  float** gpu_values = reinterpret_cast<float**>(buf_grad_value->ptr());
  int64_t* gpu_len = reinterpret_cast<int64_t*>(buf_length->ptr());
  int* d_slot_vector = reinterpret_cast<int*>(buf_slot_vector->ptr());

  cudaMemcpy(gpu_values, grad_values.data(),
             grad_values.size() * sizeof(float*), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_len, slot_lengths_lod.data(),
             slot_lengths.size() * sizeof(int64_t), cudaMemcpyHostToDevice);
  cudaMemcpy(d_slot_vector, slot_vector_.data(),
             slot_lengths_lod.size() * sizeof(int), cudaMemcpyHostToDevice);

  PushCopy<<<(total_length + 512 - 1) / 512, 512, 0, stream>>>(
      total_grad_values_gpu, gpu_values, gpu_len, hidden_size,
      slot_lengths.size(), total_length, batch_size, d_slot_vector);
  cudaStreamSynchronize(stream);
}
}  // end namespace framework
}  // end namespace paddle
#endif
