
#include <hip/hip_runtime.h>
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_BOX_PS
#include <algorithm>
#include <ctime>
#include <memory>
#include <numeric>
#include "paddle/fluid/framework/fleet/box_wrapper.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace framework {

template <size_t EMBEDX_DIM, size_t EXPAND_EMBED_DIM>
__global__ void PullCopy(
    float** dest,
    const boxps::FeatureValueGpu<EMBEDX_DIM, EXPAND_EMBED_DIM>* src,
    const int64_t* len, int hidden, int expand_dim, int slot_num, int total_len,
    uint64_t** keys) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    if (*(keys[x] + y) == 0) {
      *(dest[x] + y * hidden) = 0;
      *(dest[x] + y * hidden + 1) = 0;
      *(dest[x] + y * hidden + 2) = 0;
    } else {
      *(dest[x] + y * hidden) = (src + i)->show;
      *(dest[x] + y * hidden + 1) = (src + i)->clk;
      *(dest[x] + y * hidden + 2) = (src + i)->embed_w;
    }
    if ((src + i)->embedding_size == 0 || *(keys[x] + y) == 0) {
      for (int j = 0; j < hidden - 3; j++) {
        *(dest[x] + y * hidden + 3 + j) = 0;
      }
    } else {
      for (int j = 0; j < hidden - 3; j++) {
        *(dest[x] + y * hidden + 3 + j) = (src + i)->embedx[1 + j];
      }
    }
    // process embed_expand
    if (expand_dim > 0) {
      int z = x + slot_num;
      if ((src + i)->embed_expand_size[0] == 0 || *(keys[x] + y) == 0) {
        for (int j = 0; j < expand_dim; j++) {
          *(dest[z] + y * expand_dim + j) = 0;
        }
      } else {
        for (int j = 0; j < expand_dim; j++) {
          *(dest[z] + y * expand_dim + j) = (src + i)->embed_expand[1 + j];
        }
      }
    }
  }  // end kernel loop
}

__global__ void CopyKeysKernel(uint64_t** src_keys, uint64_t* dest_total_keys,
                               const int64_t* len, int slot_num,
                               int total_len) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    dest_total_keys[i] = src_keys[x][y];
  }
}

template <size_t EMBEDX_DIM, size_t EXPAND_EMBED_DIM>
__global__ void PushCopy(
    boxps::FeaturePushValueGpu<EMBEDX_DIM, EXPAND_EMBED_DIM>* dest, float** src,
    int64_t* len, int hidden, int expand_dim, int slot_num, int total_len,
    int bs, int* slot_vector) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[low - 1] : 0);
    (dest + i)->slot = slot_vector[x];
    (dest + i)->show = *(src[x] + y * hidden);
    (dest + i)->clk = *(src[x] + y * hidden + 1);
    (dest + i)->embed_g = *(src[x] + y * hidden + 2) * -1. * bs;
    for (int j = 0; j < hidden - 3; j++) {
      (dest + i)->embedx_g[j] = *(src[x] + y * hidden + 3 + j) * -1. * bs;
    }
    if (expand_dim > 0) {
      int z = x + slot_num;
      for (int j = 0; j < expand_dim; j++) {
        (dest + i)->embed_expand_g[j] =
            *(src[z] + y * expand_dim + j) * -1. * bs;
      }
    }
  }
}

void BoxWrapper::CopyForPull(const paddle::platform::Place& place,
                             uint64_t** gpu_keys,
                             const std::vector<float*>& values,
                             void* total_values_gpu, const int64_t* gpu_len,
                             const int slot_num, const int hidden_size,
                             const int expand_embed_dim,
                             const int64_t total_length) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  auto buf_value = memory::AllocShared(place, values.size() * sizeof(float*));
  float** gpu_values = reinterpret_cast<float**>(buf_value->ptr());
#ifdef PADDLE_WITH_HIP
  hipMemcpy(gpu_values, values.data(), values.size() * sizeof(float*),
            hipMemcpyHostToDevice);
#else
  cudaMemcpy(gpu_values, values.data(), values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);
#endif
#define EMBEDX_CASE(i, ...)                                                  \
  case i: {                                                                  \
    constexpr size_t EmbedxDim = i;                                          \
    switch (expand_embed_dim) {                                              \
      __VA_ARGS__                                                            \
      default:                                                               \
        PADDLE_THROW(platform::errors::InvalidArgument(                      \
            "Unsupport this expand embedding size [%d]", expand_embed_dim)); \
    }                                                                        \
  } break

#ifdef PADDLE_WITH_HIP
#define EXPAND_EMBED_PUSH_CASE(i, ...)                                        \
  case i: {                                                                   \
    constexpr size_t ExpandDim = i;                                           \
    hipLaunchKernelGGL(                                                       \
        PushCopy<EmbedxDim, ExpandDim>, dim3((total_length + 512 - 1) / 512), \
        dim3(512), 0, stream, gpu_values,                                     \
        reinterpret_cast<boxps::FeatureValueGpu<EmbedxDim, ExpandDim>*>(      \
            total_values_gpu),                                                \
        gpu_len, hidden_size, expand_embed_dim, slot_num, total_length,       \
        gpu_keys);                                                            \
  } break
#else
#define EXPAND_EMBED_PULL_CASE(i, ...)                                       \
  case i: {                                                                  \
    constexpr size_t ExpandDim = i;                                          \
    PullCopy<EmbedxDim,                                                      \
             ExpandDim><<<(total_length + 512 - 1) / 512, 512, 0, stream>>>( \
        gpu_values,                                                          \
        reinterpret_cast<boxps::FeatureValueGpu<EmbedxDim, ExpandDim>*>(     \
            total_values_gpu),                                               \
        gpu_len, hidden_size, expand_embed_dim, slot_num, total_length,      \
        gpu_keys);                                                           \
  } break
#endif

  switch (hidden_size - 3) {
    EMBEDX_CASE(8, EXPAND_EMBED_PULL_CASE(0); EXPAND_EMBED_PULL_CASE(8);
                EXPAND_EMBED_PULL_CASE(64););
    EMBEDX_CASE(16, EXPAND_EMBED_PULL_CASE(0););
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupport this embedding size [%d]", hidden_size - 3));
  }
  cudaStreamSynchronize(stream);
#undef EXPAND_EMBED_PULL_CASE
#undef EMBEDX_CASE
}

void BoxWrapper::CopyKeys(const paddle::platform::Place& place,
                          uint64_t** origin_keys, uint64_t* total_keys,
                          const int64_t* gpu_len, int slot_num, int total_len) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
#ifdef PADDLE_WITH_HIP
  hipLaunchKernelGGL(CopyKeysKernel, dim3((total_len + 512 - 1) / 512),
                     dim3(512), 0, stream, origin_keys, total_keys, gpu_len,
                     slot_num, total_len);
  hipStreamSynchronize(stream);
#else
  CopyKeysKernel<<<(total_len + 512 - 1) / 512, 512, 0, stream>>>(
      origin_keys, total_keys, gpu_len, slot_num, total_len);
  cudaStreamSynchronize(stream);
#endif
}

void BoxWrapper::CopyForPush(const paddle::platform::Place& place,
                             const std::vector<const float*>& grad_values,
                             void* total_grad_values_gpu,
                             const std::vector<int64_t>& slot_lengths,
                             const int hidden_size, const int expand_embed_dim,
                             const int64_t total_length, const int batch_size) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  auto slot_lengths_lod = slot_lengths;
  for (int i = 1; i < slot_lengths_lod.size(); i++) {
    slot_lengths_lod[i] += slot_lengths_lod[i - 1];
  }
  auto buf_grad_value =
      memory::AllocShared(place, grad_values.size() * sizeof(float*));
  auto buf_length =
      memory::AllocShared(place, slot_lengths.size() * sizeof(int64_t));
  auto buf_slot_vector =
      memory::AllocShared(place, slot_lengths_lod.size() * sizeof(int));

  float** gpu_values = reinterpret_cast<float**>(buf_grad_value->ptr());
  int64_t* gpu_len = reinterpret_cast<int64_t*>(buf_length->ptr());
  int* d_slot_vector = reinterpret_cast<int*>(buf_slot_vector->ptr());

#ifdef PADDLE_WITH_HIP
  hipMemcpy(gpu_values, grad_values.data(), grad_values.size() * sizeof(float*),
            hipMemcpyHostToDevice);
  hipMemcpy(gpu_len, slot_lengths_lod.data(),
            slot_lengths.size() * sizeof(int64_t), hipMemcpyHostToDevice);
  hipMemcpy(d_slot_vector, slot_vector_.data(),
            slot_lengths_lod.size() * sizeof(int), hipMemcpyHostToDevice);
#else
  cudaMemcpy(gpu_values, grad_values.data(),
             grad_values.size() * sizeof(float*), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_len, slot_lengths_lod.data(),
             slot_lengths.size() * sizeof(int64_t), cudaMemcpyHostToDevice);
  cudaMemcpy(d_slot_vector, slot_vector_.data(),
             slot_lengths_lod.size() * sizeof(int), cudaMemcpyHostToDevice);
#endif

#define EMBEDX_CASE(i, ...)                                                  \
  case i: {                                                                  \
    constexpr size_t EmbedxDim = i;                                          \
    switch (expand_embed_dim) {                                              \
      __VA_ARGS__                                                            \
      default:                                                               \
        PADDLE_THROW(platform::errors::InvalidArgument(                      \
            "Unsupport this expand embedding size [%d]", expand_embed_dim)); \
    }                                                                        \
  } break

#ifdef PADDLE_WITH_HIP
#define EXPAND_EMBED_PUSH_CASE(i, ...)                                       \
  case i: {                                                                  \
    constexpr size_t ExpandDim = i;                                          \
    hipLaunchKernelGGL(PushCopy<EmbedxDim, ExpandDim>,                       \
        dim3(total_length + 512 - 1) / 512), dim3(512), 0, stream,           \
        reinterpret_cast<boxps::FeaturePushValueGpu<EmbedxDim, ExpandDim>*>( \
            total_grad_values_gpu),                                          \
        gpu_values, gpu_len, hidden_size, expand_embed_dim,                  \
        slot_lengths.size(), total_length, batch_size, d_slot_vector);       \
  } break
#else
#define EXPAND_EMBED_PUSH_CASE(i, ...)                                       \
  case i: {                                                                  \
    constexpr size_t ExpandDim = i;                                          \
    PushCopy<EmbedxDim,                                                      \
             ExpandDim><<<(total_length + 512 - 1) / 512, 512, 0, stream>>>( \
        reinterpret_cast<boxps::FeaturePushValueGpu<EmbedxDim, ExpandDim>*>( \
            total_grad_values_gpu),                                          \
        gpu_values, gpu_len, hidden_size, expand_embed_dim,                  \
        slot_lengths.size(), total_length, batch_size, d_slot_vector);       \
  } break
#endif

  switch (hidden_size - 3) {
    EMBEDX_CASE(8, EXPAND_EMBED_PUSH_CASE(0); EXPAND_EMBED_PUSH_CASE(8);
                EXPAND_EMBED_PUSH_CASE(64););
    EMBEDX_CASE(16, EXPAND_EMBED_PUSH_CASE(0););
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupport this embedding size [%d]", hidden_size - 3));
  }

  cudaStreamSynchronize(stream);
#undef EXPAND_EMBED_PUSH_CASE
#undef EMBEDX_CASE
}

}  // end namespace framework
}  // end namespace paddle
#endif
