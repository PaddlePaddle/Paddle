/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <hip/hip_runtime.h>


#ifdef PADDLE_WITH_HETERPS
#include "paddle/fluid/framework/fleet/heter_ps/heter_comm_kernel.h"

namespace paddle {
namespace framework {

#ifdef PADDLE_WITH_CUDA

struct GPUCustomGradMerger {
  template <typename T>
  CUB_RUNTIME_FUNCTION __forceinline__ __device__ T
  operator()(const T& a, const T& b) const {
    T out;
    out.slot = a.slot;
    out.show = a.show + b.show;
    out.clk = a.clk + b.clk;
    out.lr_g = a.lr_g + b.lr_g;
    for (int i = 0; i < MF_DIM; ++i) {
      out.mf_g[i] = a.mf_g[i] + b.mf_g[i];
    }
    return out;
  }
} gpu_merger;

template <typename T>
__global__ void fill_idx(T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    idx[i] = i;
  }
}

// template <typename T>
// void show_tensor(T* input, size_t len, gpuStream_t stream, std::string
// name)
// {
//  T tmp[len];  // NOLINT
//  cudaMemcpyAsync(&tmp, input, sizeof(T) * len, cudaMemcpyDeviceToHost,
//  stream);
//  cudaStreamSynchronize(stream);
//  std::cout << name;
//  for (int i = 0; i < len; ++i) {
//    std::cout << ":" << tmp[i];
//  }
//  std::cout << std::endl;
//}

template <typename T>
__global__ void calc_shard_offset(T* idx, T* left, T* right, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len - 1) {
    if (idx[i] != idx[i + 1]) {
      right[idx[i]] = i;
      left[idx[i + 1]] = i + 1;
    }
  }
  if (i == 0) {
    left[idx[i]] = i;
  }
  if (i == (len - 1)) {
    right[idx[i]] = i;
  }
}

template <typename KeyType, typename T>
__global__ void calc_shard_index(KeyType* d_keys, size_t len, T* shard_index,
                                 int total_gpu) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    shard_index[i] = d_keys[i] % total_gpu;
  }
}

template <typename KeyType, typename T>
__global__ void fill_shard_key(KeyType* d_shard_keys, KeyType* d_keys, T* idx,
                               size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
  }
}

template <typename KeyType, typename GradType, typename T>
__global__ void fill_shard_grads(KeyType* d_shard_keys, KeyType* d_keys,
                                 GradType* d_shard_grads, GradType* d_grads,
                                 T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
    d_shard_grads[i] = d_grads[idx[i]];
  }
}

template <typename ValType, typename T>
__global__ void fill_dvals(ValType* d_shard_vals, ValType* d_vals, T* idx,
                           size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_vals[idx[i]] = d_shard_vals[i];
  }
}

// cuda implemention of  heter_comm_kernel.h
template <typename T, typename StreamType>
void HeterCommKernel::fill_idx(T* idx, long long len,
                               const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_idx<<<grid_size, block_size_, 0, stream>>>(idx, c_len);
}

template <typename T, typename StreamType>
void HeterCommKernel::calc_shard_offset(T* idx, T* left, T* right,
                                        long long len, int total_devs,
                                        const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  calc_shard_offset<<<grid_size, block_size_, 0, stream>>>(idx, left, right,
                                                           c_len);
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::calc_shard_index(KeyType* d_keys, long long len,
                                       T* shard_index, int total_gpu,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  calc_shard_index<<<grid_size, block_size_, 0, stream>>>(
      d_keys, c_len, shard_index, total_gpu);
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::fill_shard_key(KeyType* d_shard_keys, KeyType* d_keys,
                                     T* idx, long long len,
                                     const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_shard_key<<<grid_size, block_size_, 0, stream>>>(d_shard_keys, d_keys,
                                                        idx, c_len);
}

template <typename KeyType, typename GradType, typename T, typename StreamType>
void HeterCommKernel::fill_shard_grads(KeyType* d_shard_keys, KeyType* d_keys,
                                       GradType* d_shard_grads,
                                       GradType* d_grads, T* idx, long long len,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_shard_grads<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, d_shard_grads, d_grads, idx, c_len);
}

template <typename ValType, typename T, typename StreamType>
void HeterCommKernel::fill_dvals(ValType* d_shard_vals, ValType* d_vals, T* idx,
                                 long long len, const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_dvals<<<grid_size, block_size_, 0, stream>>>(d_shard_vals, d_vals, idx,
                                                    c_len);
}

template <typename KeyT, typename ValueT, typename StreamType>
void HeterCommKernel::sort_pairs(
    void* d_temp_storage, size_t& temp_storage_bytes,  // NOLINT
    const KeyT* d_keys_in,                             // NOLINT
    KeyT* d_keys_out, const ValueT* d_values_in, ValueT* d_values_out,
    int num_items, int begin_bit = 0, int end_bit = sizeof(KeyT) * 8,
    StreamType stream = 0, bool debug_synchronous = false) {
  PADDLE_ENFORCE_GPU_SUCCESS(cub::DeviceRadixSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, begin_bit, end_bit, stream, debug_synchronous));
}

template <typename KeysInputIteratorT, typename UniqueOutputIteratorT,
          typename ValuesInputIteratorT, typename AggregatesOutputIteratorT,
          typename NumRunsOutputIteratorT, typename StreamType>
void HeterCommKernel::reduce_by_key(void* d_temp_storage,
                                    size_t& temp_storage_bytes,  // NOLINT
                                    KeysInputIteratorT d_keys_in,
                                    UniqueOutputIteratorT d_unique_out,
                                    ValuesInputIteratorT d_values_in,
                                    AggregatesOutputIteratorT d_aggregates_out,
                                    NumRunsOutputIteratorT d_num_runs_out,
                                    int num_items, StreamType stream = 0,
                                    bool debug_synchronous = false) {
  PADDLE_ENFORCE_GPU_SUCCESS(cub::DeviceReduce::ReduceByKey(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_unique_out, d_values_in,
      d_aggregates_out, d_num_runs_out, gpu_merger, num_items, stream,
      debug_synchronous));
}

template void HeterCommKernel::fill_idx<int, cudaStream_t>(
    int* idx, long long len, const cudaStream_t& stream);
template void HeterCommKernel::calc_shard_offset<int, cudaStream_t>(
    int* idx, int* left, int* right, long long len, int total_devs,
    const cudaStream_t& stream);
template void HeterCommKernel::calc_shard_index<
    unsigned long, int, cudaStream_t>(unsigned long* d_keys, long long len,
                                      int* shard_index, int total_gpu,
                                      const cudaStream_t& stream);
template void HeterCommKernel::fill_shard_key<unsigned long, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys, int* idx, long long len,
    const cudaStream_t& stream) {
  template void HeterCommKernel::fill_shard_grads<
      unsigned long, paddle::framework::FeaturePushValue, int, cudaStream_t>(
      unsigned long* d_shard_keys, unsigned long* d_keys,
      paddle::framework::FeaturePushValue* d_shard_grads,
      paddle::framework::FeaturePushValue* d_grads, int* idx, long long len,
      const cudaStream_t& stream);
  template void HeterCommKernel::fill_dvals<paddle::framework::FeatureValue,
                                            int, cudaStream_t>(
      paddle::framework::FeatureValue * d_shard_vals,
      paddle::framework::FeatureValue * d_vals, int* idx, long long len,
      const cudaStream_t& stream);
  template void HeterCommKernel::sort_pairs<
      unsigned long, paddle::framework::FeatureValue, cudaStream_t>(
      void* d_temp_storage,
      size_t& temp_storage_bytes,      // NOLINT
      const unsigned long* d_keys_in,  // NOLINT
      unsigned long* d_keys_out,
      const paddle::framework::FeatureValue* d_values_in,
      paddle::framework::FeatureValue* d_values_out, int num_items,
      int begin_bit, int end_bit, cudaStream_t stream, bool debug_synchronous);
  template void HeterCommKernel::reduce_by_key<
      unsigned long*, unsigned long*, paddle::framework::FeaturePushValue*,
      paddle::framework::FeaturePushValue*, int*, cudaStream_t>(
      void* d_temp_storage,
      size_t& temp_storage_bytes,  // NOLINT
      unsigned long* d_keys_in, unsigned long* d_unique_out,
      paddle::framework::FeaturePushValue* d_values_in,
      paddle::framework::FeaturePushValue* d_aggregates_out,
      int* d_num_runs_out, int num_items, cudaStream_t stream,
      bool debug_synchronous);

#endif
}  // namespace framework
}  // namespace paddle
#endif
