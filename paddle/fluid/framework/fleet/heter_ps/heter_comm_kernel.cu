/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <hip/hip_runtime.h>


#ifdef PADDLE_WITH_HETERPS
#include "paddle/fluid/framework/fleet/heter_ps/heter_comm_kernel.h"

namespace paddle {
namespace framework {

#ifdef PADDLE_WITH_CUDA

struct GPUCustomGradMerger {
  template <typename T>
  CUB_RUNTIME_FUNCTION __forceinline__ __device__ T
  operator()(const T& a, const T& b) const {
    T out;
    out.slot = a.slot;
    out.show = a.show + b.show;
    out.clk = a.clk + b.clk;
    out.lr_g = a.lr_g + b.lr_g;
    for (int i = 0; i < MF_DIM; ++i) {
      out.mf_g[i] = a.mf_g[i] + b.mf_g[i];
    }
    return out;
  }
} gpu_merger;

template <typename T>
__global__ void fill_idx_kernel(T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    idx[i] = i;
  }
}

// template <typename T>
// void show_tensor(T* input, size_t len, gpuStream_t stream, std::string
// name)
// {
//  T tmp[len];  // NOLINT
//  cudaMemcpyAsync(&tmp, input, sizeof(T) * len, cudaMemcpyDeviceToHost,
//  stream);
//  cudaStreamSynchronize(stream);
//  std::cout << name;
//  for (int i = 0; i < len; ++i) {
//    std::cout << ":" << tmp[i];
//  }
//  std::cout << std::endl;
//}

template <typename T>
__global__ void calc_shard_offset_kernel(T* idx, T* left, T* right,
                                         size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len - 1) {
    if (idx[i] != idx[i + 1]) {
      right[idx[i]] = i;
      left[idx[i + 1]] = i + 1;
    }
  }
  if (i == 0) {
    left[idx[i]] = i;
  }
  if (i == (len - 1)) {
    right[idx[i]] = i;
  }
}

template <typename KeyType, typename T>
__global__ void calc_shard_index_kernel(KeyType* d_keys, size_t len,
                                        T* shard_index, int total_gpu) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    shard_index[i] = d_keys[i] % total_gpu;
  }
}

template <typename KeyType, typename T>
__global__ void fill_shard_key_kernel(KeyType* d_shard_keys, KeyType* d_keys,
                                      T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
  }
}

template <typename KeyType, typename GradType, typename T>
__global__ void fill_shard_grads_kernel(KeyType* d_shard_keys, KeyType* d_keys,
                                        GradType* d_shard_grads,
                                        GradType* d_grads, T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
    d_shard_grads[i] = d_grads[idx[i]];
  }
}

template <typename ValType, typename T>
__global__ void fill_dvals_kernel(ValType* d_shard_vals, ValType* d_vals,
                                  T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_vals[idx[i]] = d_shard_vals[i];
  }
}

template <typename KeyType, typename T>
__global__ void dy_mf_fill_shard_grads_kernel(
    KeyType* d_shard_keys, KeyType* d_keys, float* d_shard_grads,
    float* d_grads, T* idx, size_t len, size_t grad_value_size,
    CommonFeatureValueAccessor feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
    float* cur = (float*)((char*)d_shard_grads + i * grad_value_size);
    float* shard_val = (float*)((char*)d_grads + uint64_t(idx[i]) * grad_value_size);

    cur[feature_value_accessor.common_push_value.SlotIndex()] =
      shard_val[feature_value_accessor.common_push_value.SlotIndex()];
    cur[feature_value_accessor.common_push_value.ShowIndex()] =
      shard_val[feature_value_accessor.common_push_value.ShowIndex()];
    cur[feature_value_accessor.common_push_value.ClickIndex()] =
      shard_val[feature_value_accessor.common_push_value.ClickIndex()];
    cur[feature_value_accessor.common_push_value.MfDimIndex()] =
      shard_val[feature_value_accessor.common_push_value.MfDimIndex()];
    cur[feature_value_accessor.common_push_value.EmbedGIndex()] =
      shard_val[feature_value_accessor.common_push_value.EmbedGIndex()];

    for (int x = 0; x < int(shard_val[feature_value_accessor.common_push_value.MfDimIndex()]); x++) {
      cur[feature_value_accessor.common_push_value.EmbedxGIndex() + x] = 
        shard_val[feature_value_accessor.common_push_value.EmbedxGIndex() + x];
    }
  }
}

template <typename KeyType>
__global__ void merge_gradients_kernel(const KeyType* d_keys,
                                       const uint32_t* offset,
                                       const uint32_t* fea_num,
                                       const uint32_t* index, const char* input,
                                       char* output, int n,
                                       size_t grad_value_size,
                                       DynamicGradMerger& merger_,
                                      CommonFeatureValueAccessor& feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    uint32_t start = offset[i];
    uint32_t num = fea_num[i];
    int ori_index = index[start];
    float* out = (float*)(output + i * grad_value_size);
    float* in =
        (float*)(input + size_t(ori_index) * grad_value_size);
    merger_.update_one(out, in, feature_value_accessor);
    KeyType key = d_keys[i];
    if (key != 0) {
      for (int j = 1; j < num; ++j) {
        ori_index = index[start + j];
        in = (float*)(input + size_t(ori_index) * grad_value_size);
        merger_.merge_one(out, in, feature_value_accessor);
      }
    }
  }
}

template <typename T>
__global__ void dy_mf_fill_dvals_kernel(float* d_shard_vals, float* d_vals,
                                        T* idx, size_t len, size_t val_size,
                                       CommonFeatureValueAccessor feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    uint64_t new_offset = uint64_t(idx[i]) * val_size;
    float* cur = (float*)((char*)d_vals + new_offset);
    float* shard_val = (float*)((char*)d_shard_vals + uint64_t(i) * val_size);
    cur[feature_value_accessor.common_feature_value.SlotIndex()] =
      shard_val[feature_value_accessor.common_feature_value.SlotIndex()];
    cur[feature_value_accessor.common_feature_value.ShowIndex()] =
      shard_val[feature_value_accessor.common_feature_value.ShowIndex()];
    cur[feature_value_accessor.common_feature_value.ClickIndex()] =
      shard_val[feature_value_accessor.common_feature_value.ClickIndex()];
    cur[feature_value_accessor.common_feature_value.MfDimIndex()] =
      shard_val[feature_value_accessor.common_feature_value.MfDimIndex()];
    cur[feature_value_accessor.common_feature_value.EmbedWIndex()] =
      shard_val[feature_value_accessor.common_feature_value.EmbedWIndex()];
    cur[feature_value_accessor.common_feature_value.MfSizeIndex()] =
      shard_val[feature_value_accessor.common_feature_value.MfSizeIndex()];
    cur[feature_value_accessor.common_feature_value.CpuPtrIndex()] =
      shard_val[feature_value_accessor.common_feature_value.CpuPtrIndex()];
    cur[feature_value_accessor.common_feature_value.DeltaScoreIndex()] =
      shard_val[feature_value_accessor.common_feature_value.DeltaScoreIndex()];
    cur[feature_value_accessor.common_feature_value.EmbedWIndex()] =
      shard_val[feature_value_accessor.common_feature_value.EmbedWIndex()];
    for (int i = 0; i < feature_value_accessor.common_feature_value.EmbedDim(); i++) {
      cur[feature_value_accessor.common_feature_value.EmbedG2SumIndex() + i] = 
        shard_val[feature_value_accessor.common_feature_value.EmbedG2SumIndex() + i];
    }

    for (int x = 0; x < feature_value_accessor.common_feature_value.EmbedXDim(); x++) {
      cur[feature_value_accessor.common_feature_value.EmbedxG2SumIndex() + x]  = 
        shard_val[feature_value_accessor.common_feature_value.EmbedxG2SumIndex() + x];
    }
    for (int x = 0; x < feature_value_accessor.common_feature_value.EmbedWDim(); x++) {
      cur[feature_value_accessor.common_feature_value.EmbedxWIndex() + x] = 
        shard_val[feature_value_accessor.common_feature_value.EmbedxWIndex() + x];
    }
  }
}

// cuda implemention of  heter_comm_kernel.h
template <typename T, typename StreamType>
void HeterCommKernel::fill_idx(T* idx, long long len,
                               const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_idx_kernel<<<grid_size, block_size_, 0, stream>>>(idx, c_len);
}

template <typename T, typename StreamType>
void HeterCommKernel::calc_shard_offset(T* idx, T* left, T* right,
                                        long long len, int total_devs,
                                        const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  calc_shard_offset_kernel<<<grid_size, block_size_, 0, stream>>>(idx, left,
                                                                  right, c_len);
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::calc_shard_index(KeyType* d_keys, long long len,
                                       T* shard_index, int total_gpu,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  calc_shard_index_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_keys, c_len, shard_index, total_gpu);
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::fill_shard_key(KeyType* d_shard_keys, KeyType* d_keys,
                                     T* idx, long long len,
                                     const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_shard_key_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, idx, c_len);
}

template <typename KeyType, typename GradType, typename T, typename StreamType>
void HeterCommKernel::fill_shard_grads(KeyType* d_shard_keys, KeyType* d_keys,
                                       GradType* d_shard_grads,
                                       GradType* d_grads, T* idx, long long len,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_shard_grads_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, d_shard_grads, d_grads, idx, c_len);
}

template <typename ValType, typename T, typename StreamType>
void HeterCommKernel::fill_dvals(ValType* d_shard_vals, ValType* d_vals, T* idx,
                                 long long len, const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_dvals_kernel<<<grid_size, block_size_, 0, stream>>>(d_shard_vals, d_vals,
                                                           idx, c_len);
}

template <typename KeyT, typename ValueT, typename StreamType>
void HeterCommKernel::sort_pairs(void* d_temp_storage,
                                 size_t& temp_storage_bytes,  // NOLINT
                                 const KeyT* d_keys_in,       // NOLINT
                                 KeyT* d_keys_out, const ValueT* d_values_in,
                                 ValueT* d_values_out, int num_items,
                                 int begin_bit, int end_bit, StreamType stream,
                                 bool debug_synchronous) {
  PADDLE_ENFORCE_GPU_SUCCESS(cub::DeviceRadixSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, begin_bit, end_bit, stream, debug_synchronous));
}

template <typename KeysInputIteratorT, typename UniqueOutputIteratorT,
          typename ValuesInputIteratorT, typename AggregatesOutputIteratorT,
          typename NumRunsOutputIteratorT, typename StreamType>
void HeterCommKernel::reduce_by_key(void* d_temp_storage,
                                    size_t& temp_storage_bytes,  // NOLINT
                                    KeysInputIteratorT d_keys_in,
                                    UniqueOutputIteratorT d_unique_out,
                                    ValuesInputIteratorT d_values_in,
                                    AggregatesOutputIteratorT d_aggregates_out,
                                    NumRunsOutputIteratorT d_num_runs_out,
                                    int num_items, StreamType stream,
                                    bool debug_synchronous) {
  PADDLE_ENFORCE_GPU_SUCCESS(cub::DeviceReduce::ReduceByKey(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_unique_out, d_values_in,
      d_aggregates_out, d_num_runs_out, gpu_merger, num_items, stream,
      debug_synchronous));
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::dy_mf_fill_shard_grads(
    KeyType* d_shard_keys, KeyType* d_keys, float* d_shard_grads,
    float* d_grads, T* idx, long long len, size_t grad_value_size,
    const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  dy_mf_fill_shard_grads_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, d_shard_grads, d_grads, idx, c_len,
      grad_value_size, feature_value_accessor_);
}

template <typename KeyType, typename StreamType>
void HeterCommKernel::merge_gradient(
    const KeyType* d_keys,
    const uint32_t* offset, const uint32_t* fea_num, const uint32_t* index,
    const char* input, char* output, int n, size_t grad_value_size,
    DynamicGradMerger& merger_, const StreamType& stream) {
  int grid_size = (n - 1) / block_size_ + 1;
  merge_gradients_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_keys,
      offset, fea_num, index, input, output, n, grad_value_size, merger_, feature_value_accessor_);
}

template <typename T, typename StreamType>
void HeterCommKernel::dy_mf_fill_dvals(float* d_shard_vals, float* d_vals,
                                       T* idx, long long len, size_t val_size,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  dy_mf_fill_dvals_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_vals, d_vals, idx, c_len, val_size, feature_value_accessor_);
}

template void HeterCommKernel::fill_idx<int, cudaStream_t>(
    int* idx, long long len, const cudaStream_t& stream);
template void HeterCommKernel::fill_idx<uint32_t, cudaStream_t>(
    uint32_t* idx, long long len, const cudaStream_t& stream);

template void HeterCommKernel::calc_shard_offset<int, cudaStream_t>(
    int* idx, int* left, int* right, long long len, int total_devs,
    const cudaStream_t& stream);
template void HeterCommKernel::calc_shard_index<
    unsigned long, int, cudaStream_t>(unsigned long* d_keys, long long len,
                                      int* shard_index, int total_devs,
                                      const cudaStream_t& stream);

template void HeterCommKernel::calc_shard_index<long, int, cudaStream_t>(
    long* d_keys, long long len, int* shard_index, int total_devs,
    const cudaStream_t& stream);

template void HeterCommKernel::fill_shard_key<long, int, cudaStream_t>(
    long* d_shard_keys, long* d_keys, int* idx, long long len,
    const cudaStream_t& stream);

template void HeterCommKernel::fill_shard_key<unsigned long, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys, int* idx, long long len,
    const cudaStream_t& stream);

template void HeterCommKernel::fill_shard_grads<
    unsigned long, float, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys,
    float* d_shard_grads,
    float* d_grads, int* idx, long long len,
    const cudaStream_t& stream);

template void
HeterCommKernel::fill_dvals<paddle::framework::FeatureValue, int, cudaStream_t>(
    paddle::framework::FeatureValue* d_shard_vals,
    paddle::framework::FeatureValue* d_vals, int* idx, long long len,
    const cudaStream_t& stream);

template void HeterCommKernel::sort_pairs<
    unsigned long, paddle::framework::FeaturePushValue, cudaStream_t>(
    void* d_temp_storage,
    size_t& temp_storage_bytes,      // NOLINT
    const unsigned long* d_keys_in,  // NOLINT
    unsigned long* d_keys_out,
    const paddle::framework::FeaturePushValue* d_values_in,
    paddle::framework::FeaturePushValue* d_values_out, int num_items,
    int begin_bit, int end_bit, cudaStream_t stream, bool debug_synchronous);

template void HeterCommKernel::sort_pairs<int, int, cudaStream_t>(
    void* d_temp_storage,
    size_t& temp_storage_bytes,  // NOLINT
    const int* d_keys_in,        // NOLINT
    int* d_keys_out, const int* d_values_in, int* d_values_out, int num_items,
    int begin_bit, int end_bit, cudaStream_t stream, bool debug_synchronous);

template void HeterCommKernel::reduce_by_key<
    unsigned long*, unsigned long*, paddle::framework::FeaturePushValue*,
    paddle::framework::FeaturePushValue*, int*, cudaStream_t>(
    void* d_temp_storage,
    size_t& temp_storage_bytes,  // NOLINT
    unsigned long* d_keys_in, unsigned long* d_unique_out,
    paddle::framework::FeaturePushValue* d_values_in,
    paddle::framework::FeaturePushValue* d_aggregates_out, int* d_num_runs_out,
    int num_items, cudaStream_t stream, bool debug_synchronous);

template void HeterCommKernel::dy_mf_fill_shard_grads<
    unsigned long, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys,
    float* d_shard_grads, float* d_grads, int* idx, long long len,
    size_t grad_value_size, const cudaStream_t& stream);

template void HeterCommKernel::merge_gradient<uint32_t, cudaStream_t>(
    const uint32_t* d_keys,
    const uint32_t* offset, const uint32_t* fea_num, const uint32_t* index,
    const char* input, char* output, int n, size_t grad_value_size,
    DynamicGradMerger& merger_, const cudaStream_t& stream);

template void HeterCommKernel::merge_gradient<uint64_t, cudaStream_t>(
    const uint64_t* d_keys,
    const uint32_t* offset, const uint32_t* fea_num, const uint32_t* index,
    const char* input, char* output, int n, size_t grad_value_size,
    DynamicGradMerger& merger_, const cudaStream_t& stream);

template void HeterCommKernel::dy_mf_fill_dvals<int, cudaStream_t>(
    float* d_shard_vals,
    float* d_vals, int* idx, long long len,
    size_t val_size, const cudaStream_t& stream);
#endif

}  // namespace framework
}  // namespace paddle
#endif
