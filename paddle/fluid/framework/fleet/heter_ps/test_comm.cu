#include "hip/hip_runtime.h"
/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <gtest/gtest.h>
#include <vector>
#include "paddle/fluid/framework/fleet/heter_ps/feature_value.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_comm.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_resource.h"
#include "paddle/fluid/framework/fleet/heter_ps/optimizer.cuh.h"
#include "paddle/fluid/platform/cuda_device_guard.h"

using namespace paddle::framework;

TEST(TEST_FLEET, heter_comm) {
  int gpu_count = 3;
  std::vector<int> dev_ids;
  dev_ids.push_back(0);
  dev_ids.push_back(1);
  dev_ids.push_back(2);
  std::shared_ptr<HeterPsResource> resource =
      std::make_shared<HeterPsResource>(dev_ids);
  resource->enable_p2p();
  std::vector<size_t> count;
  std::vector<std::vector<FeatureKey>> keys;
  std::vector<std::vector<FeatureValue>> vals;
  count.resize(dev_ids.size(), 0);
  keys.resize(dev_ids.size());
  vals.resize(dev_ids.size());

  for (int i = 0; i < 10; i++) {
    FeatureKey key;
    FeatureValue val;
    int gpu_num = i % gpu_count;
    key = i;
    val.lr = i;
    val.lr_g2sum = val.mf_size = val.show = val.clk = val.slot = 0;
    keys[gpu_num].push_back(key);
    vals[gpu_num].push_back(val);
    count[gpu_num] += 1;
  }

  size_t size = 0;
  for (size_t i = 0; i < count.size(); ++i) {
    size = std::max(size, count[i]);
  }

  auto heter_comm =
      std::make_shared<HeterComm<FeatureKey, FeatureValue, FeaturePushValue>>(
          size, resource);
  for (int i = 0; i < gpu_count; ++i) {
    std::cout << "building table: " << i << std::endl;
    heter_comm->build_ps(i, keys[i].data(), vals[i].data(), count[i], 10, 1);
    heter_comm->show_one_table(i);
  }

  std::cout << "testing pull sparse:" << std::endl;
  paddle::platform::CUDADeviceGuard guard(0);
  FeatureKey* pull_keys;
  FeatureValue* pull_vals;
  hipMallocManaged(&pull_keys, 5 * sizeof(FeatureKey));
  hipMallocManaged(&pull_vals, 5 * sizeof(FeatureValue));

  pull_keys[0] = 2;
  pull_keys[1] = 3;
  pull_keys[2] = 9;
  pull_keys[3] = 1;
  pull_keys[4] = 6;

  heter_comm->pull_sparse(0, pull_keys, pull_vals, 5);
  for (int i = 0; i < 5; i++) {
    std::cout << pull_keys[i] << ": " << pull_vals[i] << std::endl;
  }
  hipFree(pull_keys);
  hipFree(pull_vals);

  std::cout << "testing push sparse:" << std::endl;
  Optimizer<FeatureValue, FeaturePushValue> opt;
  FeatureKey* push_keys;
  FeaturePushValue* push_vals;
  hipMallocManaged(&push_keys, 5 * sizeof(FeatureKey));
  hipMallocManaged(&push_vals, 5 * sizeof(FeaturePushValue));
  push_keys[0] = 2;
  push_keys[1] = 3;
  push_keys[2] = 9;
  push_keys[3] = 1;
  push_keys[4] = 3;
  for (int i = 0; i < 5; ++i) {
    push_vals[i].lr_g = push_keys[i] * 100;
    push_vals[i].slot = push_keys[i];
    push_vals[i].show = push_keys[i];
    push_vals[i].clk = push_keys[i];
  }
  heter_comm->push_sparse(0, push_keys, push_vals, 5, opt);
  for (int i = 0; i < gpu_count; ++i) {
    std::cout << "table " << i << ";" << std::endl;
    heter_comm->show_one_table(i);
  }

  hipFree(push_keys);
  hipFree(push_vals);
}
