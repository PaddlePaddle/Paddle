// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <algorithm>
#include <vector>
#include "paddle/fluid/framework/fleet/heter_ps/feature_value.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_ps_table.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_comm.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_resource.h"
#include "paddle/fluid/framework/fleet/heter_ps/optimizer.cuh.h"
#include "paddle/fluid/platform/cuda_device_guard.h"

using namespace paddle::framework;
void prepare_file(char file_name[], std::vector<std::string> data) {
  std::ofstream ofile;
  ofile.open(file_name);
  for (auto x : data) {
    ofile << x << std::endl;
  }

  ofile.close();
}
char edge_file_name[] = "edges.txt";
TEST(TEST_FLEET, graph_sample) {
  std::vector<std::string> edges;
  int gpu_count = 3;
  std::vector<int> dev_ids;
  dev_ids.push_back(0);
  dev_ids.push_back(1);
  dev_ids.push_back(2);

  std::shared_ptr<HeterPsResource> resource =
      std::make_shared<HeterPsResource>(dev_ids);
  resource->enable_p2p();
  GpuPsGraphTable g(resource);
  int node_count = 10;
  std::vector<std::vector<int64_t>> neighbors(node_count);
  int ind = 0;
  int64_t node_id = 0;
  // std::vector<GpuPsCommGraph> graph_list(gpu_count);
  while (ind < node_count) {
    int neighbor_size = ind + 1;
    while (neighbor_size--) {
      edges.push_back(std::to_string(ind) + "\t" + std::to_string(node_id) +
                      "\t1.0");
      node_id++;
    }
    ind++;
  }
  /*
  gpu 0:
  0,3,6,9
  gpu 1:
  1,4,7
  gpu 2:
  2,5,8

  query(2,6) returns nodes [6,9,1,4,7,2]
  */
  ::paddle::distributed::GraphParameter table_proto;
  table_proto.set_gpups_mode(true);
  table_proto.set_gpups_mode_shard_num(127);
  table_proto.set_gpu_num(3);
  table_proto.set_gpups_graph_sample_class("BasicBfsGraphSampler");
  table_proto.set_gpups_graph_sample_args("5,5,1,1");
  prepare_file(edge_file_name, edges);
  g.init_cpu_table(table_proto);
  g.load(std::string(edge_file_name), std::string("e>"));
  /*
   node x's neighbor list = [(1+x)*x/2,(1+x)*x/2 + 1,.....,(1+x)*x/2 + x]
   so node 6's neighbors are [21,22...,27]
   node 7's neighbors are [28,29,..35]
    node 0's neighbors are [0]
   query([7,0,6],sample_size=3) should return [28,29,30,0,x,x,21,22,23]
   6 --index-->2
   0 --index--->0
   7 --index-->2
  */
  int64_t cpu_key[3] = {7, 0, 6};
  void *key;
  hipMalloc((void **)&key, 3 * sizeof(int64_t));
  hipMemcpy(key, cpu_key, 3 * sizeof(int64_t), hipMemcpyHostToDevice);
  auto neighbor_sample_res = g.graph_neighbor_sample(0, (int64_t *)key, 3, 3);
  int64_t *res = new int64_t[9];
  hipMemcpy(res, neighbor_sample_res->val, 72, hipMemcpyDeviceToHost);
  std::sort(res, res + 3);
  std::sort(res + 6, res + 9);
  int64_t expected_sample_val[] = {28, 29, 30, 0, -1, -1, 21, 22, 23};
  for (int i = 0; i < 9; i++) {
    if (expected_sample_val[i] != -1) {
      ASSERT_EQ(res[i], expected_sample_val[i]);
    }
  }
  delete[] res;
  delete neighbor_sample_res;
}
