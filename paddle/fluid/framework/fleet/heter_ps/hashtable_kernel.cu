
#include <hip/hip_runtime.h>
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_HETERPS
#include <thread>

#include "paddle/fluid/framework/fleet/heter_ps/hashtable.h"
#include "paddle/fluid/framework/fleet/heter_ps/optimizer.cuh.h"
namespace paddle {
namespace framework {

#if defined(PADDLE_WITH_CUDA)

template <typename value_type>
struct ReplaceOp {
  __host__ __device__ value_type operator()(value_type new_value,
                                            value_type old_value) {
    return new_value;
  }
};

template <typename Table>
__global__ void insert_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              size_t len,
                              int dft_val,
                              uint64_t* global_num) {
  ReplaceOp<typename Table::mapped_type> op;
  thrust::pair<typename Table::key_type, typename Table::mapped_type> kv;

  __shared__ uint64_t local_num;

  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();

  if (i < len) {
    kv.first = keys[i];
    kv.second = dft_val;  // fake value
    if (kv.first == 0) {
      printf("insert dft 0 key, rank=%u\n", kv.second);
    }
    auto it = table->insert(kv, op, &local_num);
    assert(it != table->end() && "error: insert fails: table is full");
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(global_num, local_num);
  }
}

template <typename Table>
__global__ void insert_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              const typename Table::mapped_type* const vals,
                              size_t len,
                              uint64_t* global_num) {
  ReplaceOp<typename Table::mapped_type> op;
  thrust::pair<typename Table::key_type, typename Table::mapped_type> kv;

  __shared__ uint64_t local_num;

  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();

  if (i < len) {
    kv.first = keys[i];
    kv.second = vals[i];
    if (kv.first == 0) {
      printf("insert 0 key, rank=%u\n", kv.second);
    }
    /*
    auto real = kv.second;
    auto expect = (kv.first / 8) % 2;
    PADDLE_ENFORCE(real == expect, "error, real:%u, expect:%u", real, expect);
    */
    auto it = table->insert(kv, op, &local_num);
    assert(it != table->end() && "error: insert fails: table is full");
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    atomicAdd(global_num, local_num);
  }
}

template <typename Table>
__global__ void insert_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              const typename Table::mapped_type* const vals,
                              size_t len) {
  ReplaceOp<typename Table::mapped_type> op;
  thrust::pair<typename Table::key_type, typename Table::mapped_type> kv;
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    kv.first = keys[i];
    kv.second = vals[i];
    auto it = table->insert(kv, op);
    assert(it != table->end() && "error: insert fails: table is full");
  }
}

template <typename Table>
__global__ void insert_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              size_t len,
                              char* pool,
                              size_t feature_value_size,
                              int start_index) {
  ReplaceOp<typename Table::mapped_type> op;
  thrust::pair<typename Table::key_type, typename Table::mapped_type> kv;

  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < len) {
    kv.first = keys[i];
    uint64_t offset = uint64_t(start_index + i) * feature_value_size;
    kv.second = (Table::mapped_type)(pool + offset);
    auto it = table->insert(kv, op);
    if (it == table->end()) {
      printf("error: insert fails: table is full");
    }
  }
}

template <typename Table>
__global__ void search_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              typename Table::mapped_type* const vals,
                              size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      vals[i] = it->second;
    }
  }
}

template <typename Table>
__global__ void search_ranks_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              typename Table::mapped_type* const vals,
                              size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      vals[i] = it->second;
      /*
      auto real = vals[i];
      auto expect = (keys[i] / 8) % 2;
      PADDLE_ENFORCE(real == expect,
              "error, real:%u, expect:%u", real, expect);
      */
    }
  }
}

template <typename Table, typename GPUAccessor>
__global__ void dy_mf_search_kernel_fill(
    Table* table,
    const typename Table::key_type* const keys,
    char* vals,
    size_t len,
    size_t pull_feature_value_size,
    GPUAccessor gpu_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      uint64_t offset = i * pull_feature_value_size;
      float* cur = reinterpret_cast<float*>(vals + offset);
      float* input = it->second;
      gpu_accessor.PullValueFill(cur, input);
    } else {
      float* cur = reinterpret_cast<float*>(&vals[i * pull_feature_value_size]);
      gpu_accessor.PullZeroValue(cur);
    }
  }
}

template <typename Table, typename GPUAccessor>
__global__ void dy_mf_search_kernel(Table* table,
                                    const typename Table::key_type* const keys,
                                    char* vals,
                                    size_t len,
                                    size_t pull_feature_value_size,
                                    GPUAccessor gpu_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      uint64_t offset = i * pull_feature_value_size;
      float* cur = reinterpret_cast<float*>(vals + offset);
      float* input = it->second;
      gpu_accessor.PullValueFill(cur, input);
    } else {
      printf("warning: pull miss key: %lu", keys[i]);
    }
  }
}

template <typename Table, typename GradType, typename Sgd>
__global__ void update_kernel(Table* table,
                              const OptimizerConfig& optimizer_config,
                              const typename Table::key_type* const keys,
                              const GradType* const grads,
                              size_t len,
                              Sgd sgd) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      sgd.update_value(optimizer_config, (it.getter())->second, grads[i]);
    }
  }
}

template <typename Table, typename Sgd>
__global__ void dy_mf_update_kernel(Table* table,
                                    const OptimizerConfig& optimizer_config,
                                    const typename Table::key_type* const keys,
                                    const char* const grads,
                                    size_t len,
                                    Sgd sgd,
                                    size_t grad_value_size) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      const float* cur =
          reinterpret_cast<const float*>(grads + i * grad_value_size);
      sgd.dy_mf_update_value(optimizer_config, (it.getter())->second, cur);
    } else {
      printf("warning: push miss key: %lu", keys[i]);
    }
  }
}

template <typename Table>
__global__ void get_keys_kernel(Table* table,
                                typename Table::key_type* d_out,
                                uint64_t* global_cursor,
                                uint64_t unused_key) {
  extern __shared__ typename Table::key_type local_key[];
  __shared__ uint64_t local_num;
  __shared__ uint64_t global_num;

  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();
  uint64_t len = table->size();
  if (idx < len) {
    typename Table::value_type val = *(table->data() + idx);
    if (val.first != unused_key) {
      uint64_t dst = atomicAdd(&local_num, 1);
      local_key[dst] = val.first;
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    global_num = atomicAdd(global_cursor, local_num);
  }
  __syncthreads();

  if (threadIdx.x < local_num) {
    d_out[global_num + threadIdx.x] = local_key[threadIdx.x];
  }
}

template <typename Table>
__global__ void get_key_values_kernel(Table* table,
                                typename Table::key_type* d_keys,
                                typename Table::mapped_type* d_vals,
                                uint64_t* global_cursor,
                                uint64_t unused_key) {
  __shared__ typename Table::key_type local_key[256];
  //__shared__ typename Table::mapped_type local_val[256];
  __shared__ uint8_t local_val[256];
  __shared__ uint64_t local_num;
  __shared__ uint64_t global_num;

  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();
  uint64_t len = table->size();
  if (idx < len) {
    typename Table::value_type val = *(table->data() + idx);
    if (val.first != unused_key) {
      uint64_t dst = atomicAdd(&local_num, 1);
      local_key[dst] = val.first;
      local_val[dst] = val.second;
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    global_num = atomicAdd(global_cursor, local_num);
  }
  __syncthreads();

  if (threadIdx.x < local_num) {
    d_keys[global_num + threadIdx.x] = local_key[threadIdx.x];
    d_vals[global_num + threadIdx.x] = local_val[threadIdx.x];
    /*
    uint32_t real = d_vals[global_num + threadIdx.x];
    uint32_t expect = (d_keys[global_num + threadIdx.x] / 8) % 2;
    PADDLE_ENFORCE(real == expect,
            "error, key:%lu real:%u expect:%u",
            local_key[threadIdx.x], real, expect);
    */
  }
}

template <typename KeyType, typename ValType>
HashTable<KeyType, ValType>::HashTable(size_t capacity) {
  container_ = new TableContainer<KeyType, ValType>(capacity);
  CUDA_RT_CALL(cudaMalloc(&device_optimizer_config_, sizeof(OptimizerConfig)));
  CUDA_RT_CALL(cudaMemcpy(device_optimizer_config_,
                          &host_optimizer_config_,
                          sizeof(OptimizerConfig),
                          cudaMemcpyHostToDevice));
  rwlock_.reset(new phi::RWLock);
}

template <typename KeyType, typename ValType>
HashTable<KeyType, ValType>::~HashTable() {
  delete container_;
  cudaFree(device_optimizer_config_);
}

template <typename KeyType, typename ValType>
void HashTable<KeyType, ValType>::set_sparse_sgd(
    const OptimizerConfig& optimizer_config) {
  host_optimizer_config_.set_sparse_sgd(optimizer_config);
  cudaMemcpy(device_optimizer_config_,
             &host_optimizer_config_,
             sizeof(OptimizerConfig),
             cudaMemcpyHostToDevice,
             stream_);
  cudaStreamSynchronize(stream_);
}

template <typename KeyType, typename ValType>
void HashTable<KeyType, ValType>::set_embedx_sgd(
    const OptimizerConfig& optimizer_config) {
  host_optimizer_config_.set_embedx_sgd(optimizer_config);
  cudaMemcpy(device_optimizer_config_,
             &host_optimizer_config_,
             sizeof(OptimizerConfig),
             cudaMemcpyHostToDevice,
             stream_);
  cudaStreamSynchronize(stream_);
}

template <typename KeyType, typename ValType>
void HashTable<KeyType, ValType>::show() {
  container_->print();
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::get(const KeyType* d_keys,
                                      ValType* d_vals,
                                      size_t len,
                                      StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  search_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len);
}

template <typename KeyType, typename ValType>
template <typename StreamType, typename GPUAccessor>
void HashTable<KeyType, ValType>::get(const KeyType* d_keys,
                                      char* d_vals,
                                      size_t len,
                                      StreamType stream,
                                      const GPUAccessor& fv_accessor) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  // infer need zero fill
  if (infer_mode_) {
    dy_mf_search_kernel_fill<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
        container_, d_keys, d_vals, len, pull_feature_value_size_, fv_accessor);
  } else {
    dy_mf_search_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
        container_, d_keys, d_vals, len, pull_feature_value_size_, fv_accessor);
  }
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::get_ranks(const KeyType* d_keys,
                                      ValType* d_vals,
                                      size_t len,
                                      StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  search_ranks_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::insert(const KeyType* d_keys,
                                         size_t len,
                                         uint64_t* global_num,
                                         int dft_val,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  insert_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, len, dft_val, global_num);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::insert(const KeyType* d_keys,
                                         const ValType* d_vals,
                                         size_t len,
                                         uint64_t* global_num,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  insert_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len, global_num);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::insert(const KeyType* d_keys,
                                         const ValType* d_vals,
                                         size_t len,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  insert_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::get_keys(KeyType* d_out,
                                           uint64_t* global_cursor,
                                           StreamType stream) {
  size_t len = container_->size();
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  KeyType unuse_key = std::numeric_limits<KeyType>::max();
  size_t shared_mem_size = sizeof(KeyType) * BLOCK_SIZE_;
  get_keys_kernel<<<grid_size, BLOCK_SIZE_, shared_mem_size, stream>>>(
      container_, d_out, global_cursor, unuse_key);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::get_key_values(KeyType* d_keys,
                                                 ValType* d_vals,
                                                 uint64_t* global_cursor,
                                                 StreamType stream) {
  const int BLOCK_SIZE = 128;
  size_t len = container_->size();
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  KeyType unuse_key = std::numeric_limits<KeyType>::max();
  size_t shared_mem_size = (sizeof(KeyType) + sizeof(ValType)) * BLOCK_SIZE_;
  get_key_values_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, global_cursor, unuse_key);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::insert(const KeyType* d_keys,
                                         size_t len,
                                         char* pool,
                                         size_t feature_value_size,
                                         size_t start_index,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  if (pool == NULL) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  insert_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, len, pool, feature_value_size, start_index);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::dump_to_cpu(int devid, StreamType stream) {
  container_->prefetch(cudaCpuDeviceId, stream);
}

template <typename KeyType, typename ValType>
template <typename Sgd, typename StreamType>
void HashTable<KeyType, ValType>::update(const KeyType* d_keys,
                                         const float* d_grads,
                                         size_t len,
                                         Sgd sgd,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  update_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, *device_optimizer_config_, d_keys, d_grads, len, sgd);
}

template <typename KeyType, typename ValType>
template <typename Sgd, typename StreamType>
void HashTable<KeyType, ValType>::update(const KeyType* d_keys,
                                         const char* d_grads,
                                         size_t len,
                                         Sgd sgd,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  dy_mf_update_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_,
      *device_optimizer_config_,
      d_keys,
      d_grads,
      len,
      sgd,
      push_grad_value_size_);
}

template class HashTable<uint64_t, float>;
template class HashTable<uint64_t, float*>;
template class HashTable<int64_t, int>;
template class HashTable<uint64_t, int>;
template class HashTable<uint64_t, uint64_t>;
template class HashTable<uint64_t, uint64_t*>;
template class HashTable<uint64_t, int64_t>;
template class HashTable<uint64_t, int64_t*>;
template class HashTable<uint64_t, uint32_t*>;
template class HashTable<uint64_t, uint32_t>;
template class HashTable<int64_t, int64_t>;
template class HashTable<int64_t, uint64_t>;
template class HashTable<int64_t, unsigned int>;

template void HashTable<uint64_t, float>::get<cudaStream_t>(
    const uint64_t* d_keys, float* d_vals, size_t len, cudaStream_t stream);

template void
HashTable<uint64_t, float*>::get<cudaStream_t, CommonFeatureValueAccessor>(
    const uint64_t* d_keys,
    char* d_vals,
    size_t len,
    cudaStream_t stream,
    const CommonFeatureValueAccessor& fv_accessor);

template void HashTable<int64_t, int>::get<cudaStream_t>(const int64_t* d_keys,
                                                         int* d_vals,
                                                         size_t len,
                                                         cudaStream_t stream);

template void HashTable<uint64_t, int>::get<cudaStream_t>(
    const uint64_t* d_keys, int* d_vals, size_t len, cudaStream_t stream);
template void HashTable<uint64_t, unsigned int>::get<cudaStream_t>(
    const uint64_t* d_keys, unsigned int* d_vals, size_t len, cudaStream_t stream);
template void HashTable<uint64_t, uint64_t>::get<cudaStream_t>(
    const uint64_t* d_keys, uint64_t* d_vals, size_t len, cudaStream_t stream);
template void HashTable<uint64_t, int64_t>::get<cudaStream_t>(
    const uint64_t* d_keys, int64_t* d_vals, size_t len, cudaStream_t stream);
template void HashTable<int64_t, uint64_t>::get<cudaStream_t>(
    const int64_t* d_keys, uint64_t* d_vals, size_t len, cudaStream_t stream);
template void HashTable<int64_t, int64_t>::get<cudaStream_t>(
    const int64_t* d_keys, int64_t* d_vals, size_t len, cudaStream_t stream);
template void HashTable<int64_t, unsigned int>::get<cudaStream_t>(
    const int64_t* d_keys,
    unsigned int* d_vals,
    size_t len,
    cudaStream_t stream);
template void HashTable<uint64_t, unsigned int>::get_ranks<cudaStream_t>(
    const uint64_t* d_keys,
    unsigned int* d_vals,
    size_t len,
    cudaStream_t stream);
// template void
// HashTable<uint64_t, paddle::framework::FeatureValue>::get<cudaStream_t>(
//    const uint64_t* d_keys, char* d_vals, size_t len, cudaStream_t
//    stream);
template void HashTable<uint64_t, float>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    const float* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<uint64_t, float*>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    size_t len,
    char* pool,
    size_t feature_value_size,
    size_t start_index,
    cudaStream_t stream);

template void HashTable<int64_t, int>::insert<cudaStream_t>(
    const int64_t* d_keys, const int* d_vals, size_t len, cudaStream_t stream);

template void HashTable<int64_t, int64_t>::insert<cudaStream_t>(
    const int64_t* d_keys,
    const int64_t* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<uint64_t, int>::insert<cudaStream_t>(
    const uint64_t* d_keys, const int* d_vals, size_t len, cudaStream_t stream);

template void HashTable<uint64_t, int64_t>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    const int64_t* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<int64_t, uint64_t>::insert<cudaStream_t>(
    const int64_t* d_keys,
    const uint64_t* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<int64_t, unsigned int>::insert<cudaStream_t>(
    const int64_t* d_keys,
    const unsigned int* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<uint64_t, uint64_t>::get_keys<cudaStream_t>(
    uint64_t* d_out, uint64_t* global_cursor, cudaStream_t stream);

template void HashTable<uint64_t, uint32_t>::get_keys<cudaStream_t>(
    uint64_t* d_out, uint64_t* global_cursor, cudaStream_t stream);

template void HashTable<uint64_t, uint32_t>::get_key_values<cudaStream_t>(
    uint64_t* d_keys, uint32_t* d_vals, uint64_t* global_cursor, cudaStream_t stream);

template void HashTable<uint64_t, uint64_t>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    uint64_t len,
    uint64_t* global_num,
    int dft_val,
    cudaStream_t stream);

template void HashTable<uint64_t, uint32_t>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    uint64_t len,
    uint64_t* global_num,
    int dft_val,
    cudaStream_t stream);

template void HashTable<uint64_t, uint32_t>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    const uint32_t* d_values,
    uint64_t len,
    uint64_t* global_num,
    cudaStream_t stream);

template void HashTable<uint64_t, uint64_t>::insert<cudaStream_t>(
    const uint64_t* d_keys,
    const uint64_t* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<uint64_t, float*>::dump_to_cpu<cudaStream_t>(
    int devid, cudaStream_t stream);

template void HashTable<uint64_t, float*>::update<
    SparseAdagradOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const uint64_t* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdagradOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);

template void HashTable<uint64_t, float*>::update<
    SparseAdagradV2Optimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const uint64_t* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdagradV2Optimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);

template void HashTable<uint64_t, float*>::update<
    StdAdagradOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const uint64_t* d_keys,
                  const char* d_grads,
                  size_t len,
                  StdAdagradOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);
template void HashTable<uint64_t, float*>::update<
    SparseAdamOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const uint64_t* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdamOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);
template void HashTable<uint64_t, float*>::update<
    SparseAdamSharedOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const uint64_t* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdamSharedOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);

// template void HashTable<uint64_t,
// paddle::framework::FeatureValue>::update<
//    Optimizer<paddle::framework::FeatureValue,
//              paddle::framework::FeaturePushValue>,
//    cudaStream_t>(const uint64_t* d_keys, const char* d_grads, size_t
//    len,
//                  Optimizer<paddle::framework::FeatureValue,
//                            paddle::framework::FeaturePushValue>
//                      sgd,
//                  cudaStream_t stream);

#endif
}  // end namespace framework
}  // end namespace paddle
#endif
