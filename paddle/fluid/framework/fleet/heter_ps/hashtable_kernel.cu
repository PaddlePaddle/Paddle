
#include <hip/hip_runtime.h>
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef PADDLE_WITH_HETERPS
#include <thread>

#include "paddle/fluid/framework/fleet/heter_ps/hashtable.h"
#include "paddle/fluid/framework/fleet/heter_ps/optimizer.cuh.h"

namespace paddle {
namespace framework {

#if defined(PADDLE_WITH_CUDA)

template <typename value_type>
struct ReplaceOp {
  __host__ __device__ value_type operator()(value_type new_value,
                                            value_type old_value) {
    return new_value;
  }
};

template <typename Table>
__global__ void insert_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              const typename Table::mapped_type* const vals,
                              size_t len) {
  ReplaceOp<typename Table::mapped_type> op;
  thrust::pair<typename Table::key_type, typename Table::mapped_type> kv;

  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    kv.first = keys[i];
    kv.second = vals[i];
    auto it = table->insert(kv, op);
    assert(it != table->end() && "error: insert fails: table is full");
  }
}

template <typename Table>
__global__ void insert_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              size_t len,
                              char* pool,
                              size_t feature_value_size,
                              int start_index) {
  ReplaceOp<typename Table::mapped_type> op;
  thrust::pair<typename Table::key_type, typename Table::mapped_type> kv;

  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < len) {
    kv.first = keys[i];
    uint64_t offset = uint64_t(start_index + i) * feature_value_size;
    kv.second = (Table::mapped_type)(pool + offset);
    auto it = table->insert(kv, op);
    assert(it != table->end() && "error: insert fails: table is full");
  }
}

template <typename Table>
__global__ void search_kernel(Table* table,
                              const typename Table::key_type* const keys,
                              typename Table::mapped_type* const vals,
                              size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      vals[i] = it->second;
    }
  }
}

template <typename Table, typename GPUAccessor>
__global__ void dy_mf_search_kernel(Table* table,
                                    const typename Table::key_type* const keys,
                                    char* vals,
                                    size_t len,
                                    size_t pull_feature_value_size,
                                    GPUAccessor gpu_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  // return;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      uint64_t offset = i * pull_feature_value_size;
      float* cur = (float*)(vals + offset);
      float* input = it->second;
      gpu_accessor.PullValueFill(cur, input);
    }
  }
}

template <typename Table, typename GradType, typename Sgd>
__global__ void update_kernel(Table* table,
                              const OptimizerConfig& optimizer_config,
                              const typename Table::key_type* const keys,
                              const GradType* const grads,
                              size_t len,
                              Sgd sgd) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      sgd.update_value(optimizer_config, (it.getter())->second, grads[i]);
    }
  }
}

template <typename Table, typename Sgd>
__global__ void dy_mf_update_kernel(Table* table,
                                    const OptimizerConfig& optimizer_config,
                                    const typename Table::key_type* const keys,
                                    const char* const grads,
                                    size_t len,
                                    Sgd sgd,
                                    size_t grad_value_size) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    auto it = table->find(keys[i]);
    if (it != table->end()) {
      float* cur = (float*)(grads + i * grad_value_size);
      sgd.dy_mf_update_value(optimizer_config, (it.getter())->second, cur);
    } else {
      printf("warning: push miss key: %lu", keys[i]);
    }
  }
}

template <typename KeyType, typename ValType>
HashTable<KeyType, ValType>::HashTable(size_t capacity) {
  container_ = new TableContainer<KeyType, ValType>(capacity);
  CUDA_RT_CALL(
      cudaMalloc((void**)&device_optimizer_config_, sizeof(OptimizerConfig)));
  CUDA_RT_CALL(cudaMemcpy((void*)device_optimizer_config_,
                          &host_optimizer_config_,
                          sizeof(OptimizerConfig),
                          cudaMemcpyHostToDevice));
  rwlock_.reset(new phi::RWLock);
}

template <typename KeyType, typename ValType>
HashTable<KeyType, ValType>::~HashTable() {
  delete container_;
  cudaFree(device_optimizer_config_);
}

template <typename KeyType, typename ValType>
void HashTable<KeyType, ValType>::set_sparse_sgd(
    const OptimizerConfig& optimizer_config) {
  host_optimizer_config_.set_sparse_sgd(optimizer_config);
  cudaMemcpy((void*)device_optimizer_config_,
             &host_optimizer_config_,
             sizeof(OptimizerConfig),
             cudaMemcpyHostToDevice);
}

template <typename KeyType, typename ValType>
void HashTable<KeyType, ValType>::set_embedx_sgd(
    const OptimizerConfig& optimizer_config) {
  host_optimizer_config_.set_embedx_sgd(optimizer_config);
  cudaMemcpy((void*)device_optimizer_config_,
             &host_optimizer_config_,
             sizeof(OptimizerConfig),
             cudaMemcpyHostToDevice);
}

template <typename KeyType, typename ValType>
void HashTable<KeyType, ValType>::show() {
  container_->print();
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::get(const KeyType* d_keys,
                                      ValType* d_vals,
                                      size_t len,
                                      StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  search_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len);
}

template <typename KeyType, typename ValType>
template <typename StreamType, typename GPUAccessor>
void HashTable<KeyType, ValType>::get(const KeyType* d_keys,
                                      char* d_vals,
                                      size_t len,
                                      StreamType stream,
                                      GPUAccessor& fv_accessor) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  dy_mf_search_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len, pull_feature_value_size_, fv_accessor);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::insert(const KeyType* d_keys,
                                         const ValType* d_vals,
                                         size_t len,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  insert_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, d_vals, len);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::insert(const KeyType* d_keys,
                                         size_t len,
                                         char* pool,
                                         size_t feature_value_size,
                                         size_t start_index,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  if (pool == NULL) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  insert_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, d_keys, len, pool, feature_value_size, start_index);
}

template <typename KeyType, typename ValType>
template <typename StreamType>
void HashTable<KeyType, ValType>::dump_to_cpu(int devid, StreamType stream) {
  container_->prefetch(cudaCpuDeviceId, stream);
  std::vector<std::thread> threads;
  size_t num = container_->size();
  KeyType unuse_key = std::numeric_limits<KeyType>::max();
  thrust::pair<KeyType, ValType>* kv = container_->data();

  int thread_num = 8;
  int len_per_thread = num / thread_num;
  int remain = num % thread_num;
  int begin = 0;

  auto dump_func = [unuse_key, kv](int left, int right) {
    for (int i = left; i < right; i++) {
      if (kv[i].first == unuse_key) {
        continue;
      }
      ValType& gpu_val = kv[i].second;
#ifdef PADDLE_WITH_PSLIB
      auto* downpour_value =
          (paddle::ps::DownpourFixedFeatureValue*)(gpu_val.cpu_ptr);
      int downpour_value_size = downpour_value->size();
      if (gpu_val.mf_size > 0 && downpour_value_size == 7) {
        downpour_value->resize(gpu_val.mf_size + downpour_value_size);
      }
      float* cpu_val = downpour_value->data();
      // cpu_val[0] = 0;
      cpu_val[1] = gpu_val.delta_score;
      cpu_val[2] = gpu_val.show;
      cpu_val[3] = gpu_val.clk;
      cpu_val[4] = gpu_val.lr;
      cpu_val[5] = gpu_val.lr_g2sum;
      cpu_val[6] = gpu_val.slot;
      if (gpu_val.mf_size > 0) {
        for (int x = 0; x < gpu_val.mf_size; x++) {
          cpu_val[x + 7] = gpu_val.mf[x];
        }
      }
#endif
    }
  };

  for (int i = 0; i < thread_num; i++) {
    threads.push_back(std::thread(
        dump_func, begin, begin + len_per_thread + (i < remain ? 1 : 0)));
    begin += len_per_thread + (i < remain ? 1 : 0);
  }
  for (std::thread& t : threads) {
    t.join();
  }

  // container_->prefetch(devid, stream);
}

template <typename KeyType, typename ValType>
template <typename Sgd, typename StreamType>
void HashTable<KeyType, ValType>::update(const KeyType* d_keys,
                                         const float* d_grads,
                                         size_t len,
                                         Sgd sgd,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  update_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_, *device_optimizer_config_, d_keys, d_grads, len, sgd);
}

template <typename KeyType, typename ValType>
template <typename Sgd, typename StreamType>
void HashTable<KeyType, ValType>::update(const KeyType* d_keys,
                                         const char* d_grads,
                                         size_t len,
                                         Sgd sgd,
                                         StreamType stream) {
  if (len == 0) {
    return;
  }
  const int grid_size = (len - 1) / BLOCK_SIZE_ + 1;
  dy_mf_update_kernel<<<grid_size, BLOCK_SIZE_, 0, stream>>>(
      container_,
      *device_optimizer_config_,
      d_keys,
      d_grads,
      len,
      sgd,
      push_grad_value_size_);
}

template class HashTable<unsigned long, float>;
template class HashTable<unsigned long, float*>;
template class HashTable<long, int>;
template class HashTable<unsigned long, int>;
template class HashTable<unsigned long, unsigned long>;
template class HashTable<unsigned long, unsigned long*>;
template class HashTable<unsigned long, long>;
template class HashTable<unsigned long, long*>;
template class HashTable<long, long>;
template class HashTable<long, unsigned long>;
template class HashTable<long, unsigned int>;

template void HashTable<unsigned long, float>::get<cudaStream_t>(
    const unsigned long* d_keys,
    float* d_vals,
    size_t len,
    cudaStream_t stream);

template void
HashTable<unsigned long, float*>::get<cudaStream_t, CommonFeatureValueAccessor>(
    const unsigned long* d_keys,
    char* d_vals,
    size_t len,
    cudaStream_t stream,
    CommonFeatureValueAccessor& fv_accessor);

template void HashTable<long, int>::get<cudaStream_t>(const long* d_keys,
                                                      int* d_vals,
                                                      size_t len,
                                                      cudaStream_t stream);

template void HashTable<unsigned long, int>::get<cudaStream_t>(
    const unsigned long* d_keys, int* d_vals, size_t len, cudaStream_t stream);
template void HashTable<unsigned long, unsigned long>::get<cudaStream_t>(
    const unsigned long* d_keys,
    unsigned long* d_vals,
    size_t len,
    cudaStream_t stream);
template void HashTable<unsigned long, long>::get<cudaStream_t>(
    const unsigned long* d_keys, long* d_vals, size_t len, cudaStream_t stream);
template void HashTable<long, unsigned long>::get<cudaStream_t>(
    const long* d_keys, unsigned long* d_vals, size_t len, cudaStream_t stream);
template void HashTable<long, long>::get<cudaStream_t>(const long* d_keys,
                                                       long* d_vals,
                                                       size_t len,
                                                       cudaStream_t stream);
template void HashTable<long, unsigned int>::get<cudaStream_t>(
    const long* d_keys, unsigned int* d_vals, size_t len, cudaStream_t stream);
// template void
// HashTable<unsigned long, paddle::framework::FeatureValue>::get<cudaStream_t>(
//    const unsigned long* d_keys, char* d_vals, size_t len, cudaStream_t
//    stream);

template void HashTable<unsigned long, float>::insert<cudaStream_t>(
    const unsigned long* d_keys,
    const float* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<unsigned long, float*>::insert<cudaStream_t>(
    const unsigned long* d_keys,
    size_t len,
    char* pool,
    size_t feature_value_size,
    size_t start_index,
    cudaStream_t stream);

template void HashTable<long, int>::insert<cudaStream_t>(const long* d_keys,
                                                         const int* d_vals,
                                                         size_t len,
                                                         cudaStream_t stream);
template void HashTable<long, long>::insert<cudaStream_t>(const long* d_keys,
                                                          const long* d_vals,
                                                          size_t len,
                                                          cudaStream_t stream);

template void HashTable<unsigned long, int>::insert<cudaStream_t>(
    const unsigned long* d_keys,
    const int* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<unsigned long, long>::insert<cudaStream_t>(
    const unsigned long* d_keys,
    const long* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<long, unsigned long>::insert<cudaStream_t>(
    const long* d_keys,
    const unsigned long* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<long, unsigned int>::insert<cudaStream_t>(
    const long* d_keys,
    const unsigned int* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<unsigned long, unsigned long>::insert<cudaStream_t>(
    const unsigned long* d_keys,
    const unsigned long* d_vals,
    size_t len,
    cudaStream_t stream);

template void HashTable<unsigned long, float*>::dump_to_cpu<cudaStream_t>(
    int devid, cudaStream_t stream);

template void HashTable<unsigned long, float*>::update<
    SparseAdagradOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const unsigned long* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdagradOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);
template void HashTable<unsigned long, float*>::update<
    SparseAdamOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const unsigned long* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdamOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);
template void HashTable<unsigned long, float*>::update<
    SparseAdamSharedOptimizer<CommonFeatureValueAccessor>,
    cudaStream_t>(const unsigned long* d_keys,
                  const char* d_grads,
                  size_t len,
                  SparseAdamSharedOptimizer<CommonFeatureValueAccessor> sgd,
                  cudaStream_t stream);

// template void HashTable<unsigned long,
// paddle::framework::FeatureValue>::update<
//    Optimizer<paddle::framework::FeatureValue,
//              paddle::framework::FeaturePushValue>,
//    cudaStream_t>(const unsigned long* d_keys, const char* d_grads, size_t
//    len,
//                  Optimizer<paddle::framework::FeatureValue,
//                            paddle::framework::FeaturePushValue>
//                      sgd,
//                  cudaStream_t stream);

#endif
}  // end namespace framework
}  // end namespace paddle
#endif
