/* Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <hip/hip_runtime.h>


#ifdef PADDLE_WITH_HETERPS
#include "paddle/fluid/framework/fleet/heter_ps/feature_value.h"

namespace paddle {
namespace framework {

template <typename FVAccessor>
__global__ void PullCopy(float** dest,
                         const float* src,
                         const int64_t* len,
                         int slot_num,
                         int total_len,
                         uint64_t** keys,
                         uint64_t max_val_size,
                         int* gpu_dim,
                         FVAccessor feature_value_accessor) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    float* feature_value_ptr =
        static_cast<float*>((char*)src + uint64_t(i) * uint64_t(max_val_size));
    int mf_dim = gpu_dim[x] - 3;
    feature_value_accessor.Select(
        dest[x] + y * (mf_dim + 3), feature_value_ptr, keys[x] + y, mf_dim);
  }
}

template <typename FVAccessor>
__global__ void PushCopyWithPool(float* dest,
                                 float** src,
                                 int64_t* len,
                                 int slot_num,
                                 uint64_t total_len,
                                 int bs,
                                 int* slot_vector,
                                 int* mf_dim_vector,
                                 size_t grad_value_size,
                                 FVAccessor feature_value_accessor) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[low - 1] : 0);
    float* cur = static_cast<float*>((char*)dest + i * grad_value_size);

    cur[feature_value_accessor.common_push_value.SlotIndex()] =
        (float)slot_vector[x];
    int mf_dim = mf_dim_vector[x];
    cur[feature_value_accessor.common_push_value.MfDimIndex()] = mf_dim;

    cur[feature_value_accessor.common_push_value.ShowIndex()] =
        *(src[x] + y * (mf_dim + 3));
    cur[feature_value_accessor.common_push_value.ClickIndex()] =
        *(src[x] + y * (mf_dim + 3) + 1);
    cur[feature_value_accessor.common_push_value.EmbedGIndex()] =
        *(src[x] + y * (mf_dim + 3) + 2) * -1. * bs;
    for (int j = 0; j < mf_dim; j++) {
      cur[feature_value_accessor.common_push_value.EmbedxGIndex() + j] =
          *(src[x] + y * (mf_dim + 3) + 3 + j) * -1. * bs;
    }
  }
}

template <typename GPUAccessor>
void AccessorWrapper<GPUAccessor>::CopyForPullImpl(
    const paddle::platform::Place& place,
    uint64_t** gpu_keys,
    const std::vector<float*>& values,
    const float* total_values_gpu,
    const int64_t* gpu_len,
    const int slot_num,
    const int hidden_size,
    const int64_t total_length,
    int* gpu_dim,
    int feature_value_size) {
  auto stream = dynamic_cast<phi::GPUContext*>(
                    paddle::platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  auto buf_value = memory::Alloc(place, values.size() * sizeof(float*));
  float** gpu_values = reinterpret_cast<float**>(buf_value->ptr());
  cudaMemcpy(gpu_values,
             values.data(),
             values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);
  PullCopy<<<(total_length + 1024 - 1) / 1024, 1024, 0, stream>>>(
      gpu_values,
      total_values_gpu,
      gpu_len,
      slot_num,
      total_length,
      gpu_keys,
      feature_value_size,
      gpu_dim,
      gpu_accessor_);
  cudaStreamSynchronize(stream);
}

template <typename GPUAccessor>
void AccessorWrapper<GPUAccessor>::CopyForPushImpl(
    const paddle::platform::Place& place,
    const std::vector<const float*>& grad_values,
    float* total_grad_values_gpu,
    const std::vector<int64_t>& slot_lengths,
    const uint64_t total_length,
    const int batch_size,
    size_t grad_value_size,
    std::vector<int>& slot_vector,
    std::vector<int>& slot_mf_dim_vector) {
  auto stream = dynamic_cast<phi::GPUContext*>(
                    paddle::platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  auto slot_lengths_lod = slot_lengths;
  for (int i = 1; i < slot_lengths_lod.size(); i++) {
    slot_lengths_lod[i] += slot_lengths_lod[i - 1];
  }
  auto buf_grad_value =
      memory::Alloc(place, grad_values.size() * sizeof(float*));
  auto buf_length = memory::Alloc(place, slot_lengths.size() * sizeof(int64_t));
  auto buf_slot_vector =
      memory::Alloc(place, slot_lengths_lod.size() * sizeof(int));
  auto buf_mf_dim_vector =
      memory::Alloc(place, slot_lengths_lod.size() * sizeof(int));
  float** gpu_values = reinterpret_cast<float**>(buf_grad_value->ptr());
  int64_t* gpu_len = reinterpret_cast<int64_t*>(buf_length->ptr());
  int* d_slot_vector = reinterpret_cast<int*>(buf_slot_vector->ptr());
  int* d_mf_dim_vector = reinterpret_cast<int*>(buf_mf_dim_vector->ptr());
  cudaMemcpy(gpu_values,
             grad_values.data(),
             grad_values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_len,
             slot_lengths_lod.data(),
             slot_lengths.size() * sizeof(int64_t),
             cudaMemcpyHostToDevice);
  cudaMemcpy(d_slot_vector,
             slot_vector.data(),
             slot_lengths_lod.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  cudaMemcpy(d_mf_dim_vector,
             slot_mf_dim_vector.data(),
             slot_lengths_lod.size() * sizeof(int),
             cudaMemcpyHostToDevice);
  PushCopyWithPool<<<(total_length + 1024 - 1) / 1024, 1024, 0, stream>>>(
      total_grad_values_gpu,
      gpu_values,
      gpu_len,
      slot_lengths.size(),
      total_length,
      batch_size,
      d_slot_vector,
      d_mf_dim_vector,
      grad_value_size,
      gpu_accessor_);
  cudaStreamSynchronize(stream);
}

#ifdef PADDLE_WITH_PSCORE
template class AccessorWrapper<CommonFeatureValueAccessor>;
#endif

}  // namespace framework
}  // namespace paddle
#endif
