// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_wrapper.h"
#include <sstream>
<<<<<<< HEAD
=======
#include "paddle/fluid/framework/fleet/fleet_wrapper.h"
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
#include "paddle/fluid/framework/fleet/heter_ps/gpu_graph_utils.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_ps_table.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_resource.h"
DECLARE_int32(gpugraph_storage_mode);
DECLARE_bool(graph_metapath_split_opt);
namespace paddle {
namespace framework {
#ifdef PADDLE_WITH_HETERPS

std::shared_ptr<GraphGpuWrapper> GraphGpuWrapper::s_instance_(nullptr);
void GraphGpuWrapper::set_device(std::vector<int> ids) {
  for (auto device_id : ids) {
    device_id_mapping.push_back(device_id);
  }
}

<<<<<<< HEAD
=======
void GraphGpuWrapper::init_conf(const std::string &first_node_type,
                                const std::string &meta_path) {
  static std::mutex mutex;
  {
    std::lock_guard<std::mutex> lock(mutex);
    if (conf_initialized_) {
      return;
    }
    VLOG(2) << "init path config";
    conf_initialized_ = true;
    auto node_types =
        paddle::string::split_string<std::string>(first_node_type, ";");
    VLOG(2) << "node_types: " << first_node_type;
    for (auto &type : node_types) {
      auto iter = feature_to_id.find(type);
      PADDLE_ENFORCE_NE(iter,
                        feature_to_id.end(),
                        platform::errors::NotFound(
                            "(%s) is not found in feature_to_id.", type));
      VLOG(2) << "feature_to_id[" << type << "] = " << iter->second;
      first_node_type_.push_back(iter->second);
    }
    meta_path_.resize(first_node_type_.size());
    auto meta_paths = paddle::string::split_string<std::string>(meta_path, ";");

    for (size_t i = 0; i < meta_paths.size(); i++) {
      auto path = meta_paths[i];
      auto nodes = paddle::string::split_string<std::string>(path, "-");
      for (auto &node : nodes) {
        auto iter = edge_to_id.find(node);
        PADDLE_ENFORCE_NE(iter,
                          edge_to_id.end(),
                          platform::errors::NotFound(
                              "(%s) is not found in edge_to_id.", node));
        VLOG(2) << "edge_to_id[" << node << "] = " << iter->second;
        meta_path_[i].push_back(iter->second);
      }
    }
    int max_dev_id = 0;
    for (size_t i = 0; i < device_id_mapping.size(); i++) {
      if (device_id_mapping[i] > max_dev_id) {
        max_dev_id = device_id_mapping[i];
      }
    }
    finish_node_type_.resize(max_dev_id + 1);
    node_type_start_.resize(max_dev_id + 1);
    global_infer_node_type_start_.resize(max_dev_id + 1);
    for (size_t i = 0; i < device_id_mapping.size(); i++) {
      int dev_id = device_id_mapping[i];
      auto &node_type_start = node_type_start_[i];
      auto &infer_node_type_start = global_infer_node_type_start_[i];
      auto &finish_node_type = finish_node_type_[i];
      finish_node_type.clear();

      for (size_t idx = 0; idx < feature_to_id.size(); idx++) {
        infer_node_type_start[idx] = 0;
      }
      for (auto &type : node_types) {
        auto iter = feature_to_id.find(type);
        node_type_start[iter->second] = 0;
        infer_node_type_start[iter->second] = 0;
      }
      infer_cursor_.push_back(0);
      cursor_.push_back(0);
    }
    init_type_keys();
  }
}

void GraphGpuWrapper::init_type_keys() {
  size_t thread_num = device_id_mapping.size();
  int cnt = 0;

  auto &graph_all_type_total_keys = get_graph_type_keys();
  auto &type_to_index = get_graph_type_to_index();
  std::vector<std::vector<uint64_t>> tmp_keys;
  tmp_keys.resize(thread_num);
  int first_node_idx;
  d_graph_all_type_total_keys_.resize(graph_all_type_total_keys.size());
  h_graph_all_type_keys_len_.resize(graph_all_type_total_keys.size());
  for (size_t f_idx = 0; f_idx < graph_all_type_total_keys.size(); f_idx++) {
    for (size_t j = 0; j < tmp_keys.size(); j++) {
      tmp_keys[j].clear();
    }
    d_graph_all_type_total_keys_[f_idx].resize(thread_num);
    auto &type_total_key = graph_all_type_total_keys[f_idx];
    for (size_t j = 0; j < type_total_key.size(); j++) {
      uint64_t shard = type_total_key[j] % thread_num;
      tmp_keys[shard].push_back(type_total_key[j]);
    }
    for (size_t j = 0; j < thread_num; j++) {
      h_graph_all_type_keys_len_[f_idx].push_back(tmp_keys[j].size());
      VLOG(1) << "node type: " << type_to_index[f_idx]
              << ", gpu_graph_device_keys[" << j
              << "] = " << tmp_keys[j].size();
    }
    for (size_t j = 0; j < thread_num; j++) {
      auto stream = get_local_stream(j);
      int gpuid = device_id_mapping[j];
      auto place = platform::CUDAPlace(gpuid);
      platform::CUDADeviceGuard guard(gpuid);
      d_graph_all_type_total_keys_[f_idx][j] =
          memory::AllocShared(place, tmp_keys[j].size() * sizeof(uint64_t));
      hipMemcpyAsync(d_graph_all_type_total_keys_[f_idx][j]->ptr(),
                      tmp_keys[j].data(),
                      sizeof(uint64_t) * tmp_keys[j].size(),
                      hipMemcpyHostToDevice,
                      stream);
    }
  }
  for (int i = 0; i < thread_num; i++) {
    auto stream = get_local_stream(i);
    hipStreamSynchronize(stream);
  }
}

void GraphGpuWrapper::init_metapath(std::string cur_metapath,
                                    int cur_metapath_index,
                                    int cur_metapath_len) {
  cur_metapath_ = cur_metapath;
  cur_metapath_index_ = cur_metapath_index;
  cur_metapath_len_ = cur_metapath_len;

  auto nodes = paddle::string::split_string<std::string>(cur_metapath_, "-");
  cur_parse_metapath_.clear();
  cur_parse_reverse_metapath_.clear();
  for (auto &node : nodes) {
    VLOG(2) << "node: " << node << " , in metapath: " << cur_metapath_;
    auto iter = edge_to_id.find(node);
    PADDLE_ENFORCE_NE(
        iter,
        edge_to_id.end(),
        platform::errors::NotFound("(%s) is not found in edge_to_id.", node));
    cur_parse_metapath_.push_back(iter->second);
    auto etype_split = paddle::string::split_string<std::string>(node, "2");
    std::string reverse_type = etype_split[1] + "2" + etype_split[0];
    iter = edge_to_id.find(reverse_type);
    PADDLE_ENFORCE_NE(iter,
                      edge_to_id.end(),
                      platform::errors::NotFound(
                          "(%s) is not found in edge_to_id.", reverse_type));
    cur_parse_reverse_metapath_.push_back(iter->second);
  }

  size_t thread_num = device_id_mapping.size();
  cur_metapath_start_.resize(thread_num);
  for (size_t i = 0; i < thread_num; i++) {
    cur_metapath_start_[i] = 0;
  }

  auto &graph_all_type_total_keys = get_graph_type_keys();
  auto &type_to_index = get_graph_type_to_index();
  std::vector<std::vector<uint64_t>> tmp_keys;
  tmp_keys.resize(thread_num);
  int first_node_idx;
  std::string first_node =
      paddle::string::split_string<std::string>(cur_metapath_, "2")[0];
  auto it = feature_to_id.find(first_node);
  first_node_idx = it->second;
  d_graph_train_total_keys_.resize(thread_num);
  h_graph_train_keys_len_.resize(thread_num);

  for (size_t j = 0; j < tmp_keys.size(); j++) {
    tmp_keys[j].clear();
  }
  size_t f_idx = type_to_index[first_node_idx];
  auto &type_total_key = graph_all_type_total_keys[f_idx];

  VLOG(2) << "first node type:" << first_node_idx
          << ", node start size:" << type_total_key.size();

  for (size_t j = 0; j < type_total_key.size(); j++) {
    uint64_t shard = type_total_key[j] % thread_num;
    tmp_keys[shard].push_back(type_total_key[j]);
  }
  auto fleet_ptr = framework::FleetWrapper::GetInstance();
  std::shuffle(
      tmp_keys.begin(), tmp_keys.end(), fleet_ptr->LocalRandomEngine());

  for (size_t j = 0; j < thread_num; j++) {
    h_graph_train_keys_len_[j] = tmp_keys[j].size();
    VLOG(2) << j << " th card, graph train keys len: " << tmp_keys[j].size();
  }

  for (size_t j = 0; j < thread_num; j++) {
    auto stream = get_local_stream(j);
    int gpuid = device_id_mapping[j];
    auto place = platform::CUDAPlace(gpuid);
    platform::CUDADeviceGuard guard(gpuid);
    d_graph_train_total_keys_[j] =
        memory::AllocShared(place, tmp_keys[j].size() * sizeof(uint64_t));
    hipMemcpyAsync(d_graph_train_total_keys_[j]->ptr(),
                    tmp_keys[j].data(),
                    sizeof(uint64_t) * tmp_keys[j].size(),
                    hipMemcpyHostToDevice,
                    stream);
  }
}

void GraphGpuWrapper::clear_metapath_state() {
  size_t thread_num = device_id_mapping.size();
  for (size_t j = 0; j < thread_num; j++) {
    cur_metapath_start_[j] = 0;
    h_graph_train_keys_len_[j] = 0;
    d_graph_train_total_keys_[j].reset();
    for (size_t k = 0; k < cur_parse_metapath_.size(); k++) {
      ((GpuPsGraphTable *)graph_table)
          ->clear_graph_info(j, cur_parse_metapath_[k]);
    }
  }
  std::vector<int> clear_etype;
  for (size_t j = 0; j < cur_parse_metapath_.size(); j++) {
    if (find(clear_etype.begin(), clear_etype.end(), cur_parse_metapath_[j]) ==
        clear_etype.end()) {
      clear_etype.push_back(cur_parse_metapath_[j]);
    }
  }
  for (size_t j = 0; j < cur_parse_reverse_metapath_.size(); j++) {
    if (find(clear_etype.begin(),
             clear_etype.end(),
             cur_parse_reverse_metapath_[j]) == clear_etype.end()) {
      clear_etype.push_back(cur_parse_reverse_metapath_[j]);
    }
  }
  for (size_t j = 0; j < clear_etype.size(); j++) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->clear_graph(clear_etype[j]);
  }
}

>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
int GraphGpuWrapper::get_all_id(int type,
                                int slice_num,
                                std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, slice_num, output);
}

int GraphGpuWrapper::get_all_neighbor_id(
    int type, int slice_num, std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_neighbor_id(type, slice_num, output);
}

int GraphGpuWrapper::get_all_id(int type,
                                int idx,
                                int slice_num,
                                std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, idx, slice_num, output);
}

int GraphGpuWrapper::get_all_neighbor_id(
    int type,
    int idx,
    int slice_num,
    std::vector<std::vector<uint64_t>> *output) {
<<<<<<< HEAD
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_neighbor_id(type, idx, slice_num, output);
}

int GraphGpuWrapper::get_all_feature_ids(
    int type,
    int idx,
    int slice_num,
    std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_feature_ids(type, idx, slice_num, output);
}

=======
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_neighbor_id(type, idx, slice_num, output);
}

int GraphGpuWrapper::get_all_feature_ids(
    int type,
    int idx,
    int slice_num,
    std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_feature_ids(type, idx, slice_num, output);
}

int GraphGpuWrapper::get_node_embedding_ids(
    int slice_num, std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_node_embedding_ids(slice_num, output);
}

>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
void GraphGpuWrapper::set_up_types(std::vector<std::string> &edge_types,
                                   std::vector<std::string> &node_types) {
  id_to_edge = edge_types;
  for (size_t table_id = 0; table_id < edge_types.size(); table_id++) {
    int res = edge_to_id.size();
    edge_to_id[edge_types[table_id]] = res;
  }
  id_to_feature = node_types;
  for (size_t table_id = 0; table_id < node_types.size(); table_id++) {
    int res = feature_to_id.size();
    feature_to_id[node_types[table_id]] = res;
  }
  table_feat_mapping.resize(node_types.size());
  this->table_feat_conf_feat_name.resize(node_types.size());
  this->table_feat_conf_feat_dtype.resize(node_types.size());
  this->table_feat_conf_feat_shape.resize(node_types.size());
}

void GraphGpuWrapper::set_feature_separator(std::string ch) {
  feature_separator_ = ch;
  if (graph_table != nullptr) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->set_feature_separator(feature_separator_);
  }
}

<<<<<<< HEAD
=======
void GraphGpuWrapper::set_slot_feature_separator(std::string ch) {
  slot_feature_separator_ = ch;
  if (graph_table != nullptr) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->set_slot_feature_separator(slot_feature_separator_);
  }
}

>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
void GraphGpuWrapper::make_partitions(int idx,
                                      int64_t byte_size,
                                      int device_len) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->make_partitions(idx, byte_size, device_len);
}
int32_t GraphGpuWrapper::load_next_partition(int idx) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->load_next_partition(idx);
}

void GraphGpuWrapper::set_search_level(int level) {
  ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->set_search_level(level);
}

std::vector<uint64_t> GraphGpuWrapper::get_partition(int idx, int num) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_partition(idx, num);
}
int32_t GraphGpuWrapper::get_partition_num(int idx) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_partition_num(idx);
}
void GraphGpuWrapper::make_complementary_graph(int idx, int64_t byte_size) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->make_complementary_graph(idx, byte_size);
}
void GraphGpuWrapper::load_edge_file(std::string name,
                                     std::string filepath,
                                     bool reverse) {
  // 'e' means load edge
  std::string params = "e";
  if (reverse) {
    // 'e<' means load edges from $2 to $1
    params += "<" + name;
  } else {
    // 'e>' means load edges from $1 to $2
    params += ">" + name;
  }
  if (edge_to_id.find(name) != edge_to_id.end()) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->Load(std::string(filepath), params);
  }
}

void GraphGpuWrapper::load_edge_file(std::string etype2files,
                                     std::string graph_data_local_path,
                                     int part_num,
                                     bool reverse) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->parse_edge_and_load(
          etype2files, graph_data_local_path, part_num, reverse);
}

void GraphGpuWrapper::load_node_file(std::string name, std::string filepath) {
  // 'n' means load nodes and 'node_type' follows

  std::string params = "n" + name;

  if (feature_to_id.find(name) != feature_to_id.end()) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->Load(std::string(filepath), params);
  }
}

<<<<<<< HEAD
void GraphGpuWrapper::load_node_and_edge(std::string etype,
                                         std::string ntype,
                                         std::string epath,
                                         std::string npath,
=======
void GraphGpuWrapper::load_node_file(std::string ntype2files,
                                     std::string graph_data_local_path,
                                     int part_num) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->parse_node_and_load(
          ntype2files, graph_data_local_path, part_num);
}

void GraphGpuWrapper::load_node_and_edge(std::string etype2files,
                                         std::string ntype2files,
                                         std::string graph_data_local_path,
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
                                         int part_num,
                                         bool reverse) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->load_node_and_edge_file(
<<<<<<< HEAD
          etype, ntype, epath, npath, part_num, reverse);
=======
          etype2files, ntype2files, graph_data_local_path, part_num, reverse);
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
}

void GraphGpuWrapper::add_table_feat_conf(std::string table_name,
                                          std::string feat_name,
                                          std::string feat_dtype,
                                          int feat_shape) {
  if (feature_to_id.find(table_name) != feature_to_id.end()) {
    int idx = feature_to_id[table_name];
    if (table_feat_mapping[idx].find(feat_name) ==
        table_feat_mapping[idx].end()) {
      int res = (int)table_feat_mapping[idx].size();
      table_feat_mapping[idx][feat_name] = res;
    }
    int feat_idx = table_feat_mapping[idx][feat_name];
    VLOG(0) << "table_name " << table_name << " mapping id " << idx;
    VLOG(0) << " feat name " << feat_name << " feat id" << feat_idx;
    if (feat_idx < table_feat_conf_feat_name[idx].size()) {
      // overide
      table_feat_conf_feat_name[idx][feat_idx] = feat_name;
      table_feat_conf_feat_dtype[idx][feat_idx] = feat_dtype;
      table_feat_conf_feat_shape[idx][feat_idx] = feat_shape;
    } else {
      // new
      table_feat_conf_feat_name[idx].push_back(feat_name);
      table_feat_conf_feat_dtype[idx].push_back(feat_dtype);
      table_feat_conf_feat_shape[idx].push_back(feat_shape);
    }
  }
  VLOG(0) << "add conf over";
}
void GraphGpuWrapper::init_search_level(int level) { search_level = level; }

gpuStream_t GraphGpuWrapper::get_local_stream(int gpuid) {
  return ((GpuPsGraphTable *)graph_table)->get_local_stream(gpuid);
}

void GraphGpuWrapper::init_service() {
<<<<<<< HEAD
  table_proto.set_task_pool_size(24);
=======
  table_proto.set_task_pool_size(64);
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
  table_proto.set_shard_num(1000);
  table_proto.set_build_sampler_on_cpu(false);
  table_proto.set_search_level(search_level);
  table_proto.set_table_name("cpu_graph_table_");
  table_proto.set_use_cache(false);
  for (int i = 0; i < id_to_edge.size(); i++)
    table_proto.add_edge_types(id_to_edge[i]);
  for (int i = 0; i < id_to_feature.size(); i++) {
    table_proto.add_node_types(id_to_feature[i]);
    auto feat_node = id_to_feature[i];
    ::paddle::distributed::GraphFeature *g_f = table_proto.add_graph_feature();
    for (int x = 0; x < table_feat_conf_feat_name[i].size(); x++) {
      g_f->add_name(table_feat_conf_feat_name[i][x]);
      g_f->add_dtype(table_feat_conf_feat_dtype[i][x]);
      g_f->add_shape(table_feat_conf_feat_shape[i][x]);
    }
  }
  std::shared_ptr<HeterPsResource> resource =
      std::make_shared<HeterPsResource>(device_id_mapping);
  resource->enable_p2p();
<<<<<<< HEAD
  GpuPsGraphTable *g = new GpuPsGraphTable(resource, 1, id_to_edge.size());
  g->init_cpu_table(table_proto);
  g->cpu_graph_table_->set_feature_separator(feature_separator_);
  graph_table = (char *)g;
=======
  GpuPsGraphTable *g = new GpuPsGraphTable(resource, id_to_edge.size());
  size_t gpu_num = device_id_mapping.size();
  g->init_cpu_table(table_proto, gpu_num);
  g->cpu_graph_table_->set_feature_separator(feature_separator_);
  g->cpu_graph_table_->set_slot_feature_separator(slot_feature_separator_);
  graph_table = (char *)g;
  upload_num = gpu_num;
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
  upload_task_pool.reset(new ::ThreadPool(upload_num));
}

void GraphGpuWrapper::finalize() {
  ((GpuPsGraphTable *)graph_table)->show_table_collisions();
}

void GraphGpuWrapper::upload_batch(int type,
                                   int idx,
                                   int slice_num,
                                   const std::string &edge_type) {
  VLOG(0) << "begin upload edge, type[" << edge_type << "]";
<<<<<<< HEAD
=======
  auto iter = edge_to_id.find(edge_type);
  idx = iter->second;
  VLOG(2) << "cur edge: " << edge_type << ",idx: " << idx;
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
  std::vector<std::vector<uint64_t>> ids;
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, idx, slice_num, &ids);
  debug_gpu_memory_info("upload_batch node start");
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;

  for (int i = 0; i < ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, idx, this]() -> int {
      VLOG(0) << "begin make_gpu_ps_graph, node_id[" << i << "]_size["
              << ids[i].size() << "]";
      GpuPsCommGraph sub_graph =
          g->cpu_graph_table_->make_gpu_ps_graph(idx, ids[i]);
      g->build_graph_on_single_gpu(sub_graph, i, idx);
      sub_graph.release_on_cpu();
      VLOG(0) << "sub graph on gpu " << i << " is built";
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  debug_gpu_memory_info("upload_batch node end");
}

// feature table
void GraphGpuWrapper::upload_batch(int type, int slice_num, int slot_num) {
<<<<<<< HEAD
  std::vector<std::vector<uint64_t>> node_ids;
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, slice_num, &node_ids);
  debug_gpu_memory_info("upload_batch feature start");
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;
  for (int i = 0; i < node_ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, this]() -> int {
      VLOG(0) << "begin make_gpu_ps_graph_fea, node_ids[" << i << "]_size["
              << node_ids[i].size() << "]";
      GpuPsCommGraphFea sub_graph =
          g->cpu_graph_table_->make_gpu_ps_graph_fea(node_ids[i], slot_num);
      // sub_graph.display_on_cpu();
      VLOG(0) << "begin build_graph_fea_on_single_gpu, node_ids[" << i
              << "]_size[" << node_ids[i].size() << "]";
      g->build_graph_fea_on_single_gpu(sub_graph, i);
      sub_graph.release_on_cpu();
      VLOG(0) << "sub graph fea on gpu " << i << " is built";
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  // g->build_graph_from_cpu(vec);
  debug_gpu_memory_info("upload_batch feature end");
=======
  if (type == 1 &&
      (FLAGS_gpugraph_storage_mode == paddle::framework::GpuGraphStorageMode::
                                          MEM_EMB_FEATURE_AND_GPU_GRAPH ||
       FLAGS_gpugraph_storage_mode == paddle::framework::GpuGraphStorageMode::
                                          SSD_EMB_AND_MEM_FEATURE_GPU_GRAPH)) {
    return;
  }
  std::vector<std::vector<uint64_t>> node_ids;
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, slice_num, &node_ids);
  debug_gpu_memory_info("upload_batch feature start");
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;
  for (int i = 0; i < node_ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, this]() -> int {
      VLOG(0) << "begin make_gpu_ps_graph_fea, node_ids[" << i << "]_size["
              << node_ids[i].size() << "]";
      GpuPsCommGraphFea sub_graph =
          g->cpu_graph_table_->make_gpu_ps_graph_fea(i, node_ids[i], slot_num);
      // sub_graph.display_on_cpu();
      VLOG(0) << "begin build_graph_fea_on_single_gpu, node_ids[" << i
              << "]_size[" << node_ids[i].size() << "]";
      g->build_graph_fea_on_single_gpu(sub_graph, i);
      sub_graph.release_on_cpu();
      VLOG(0) << "sub graph fea on gpu " << i << " is built";
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  // g->build_graph_from_cpu(vec);
  debug_gpu_memory_info("upload_batch feature end");
}

// get sub_graph_fea
std::vector<GpuPsCommGraphFea> GraphGpuWrapper::get_sub_graph_fea(
    std::vector<std::vector<uint64_t>> &node_ids, int slot_num) {
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;
  std::vector<GpuPsCommGraphFea> sub_graph_feas(node_ids.size());
  for (int i = 0; i < node_ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, this]() -> int {
      GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
      sub_graph_feas[i] =
          g->cpu_graph_table_->make_gpu_ps_graph_fea(i, node_ids[i], slot_num);
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  return sub_graph_feas;
}

// build_gpu_graph_fea
void GraphGpuWrapper::build_gpu_graph_fea(GpuPsCommGraphFea &sub_graph_fea,
                                          int i) {
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  g->build_graph_fea_on_single_gpu(sub_graph_fea, i);
  sub_graph_fea.release_on_cpu();
  VLOG(0) << "sub graph fea on gpu " << i << " is built";
  return;
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
}

NeighborSampleResult GraphGpuWrapper::graph_neighbor_sample_v3(
    NeighborSampleQuery q, bool cpu_switch, bool compress = true) {
  return ((GpuPsGraphTable *)graph_table)
      ->graph_neighbor_sample_v3(q, cpu_switch, compress);
}

NeighborSampleResultV2 GraphGpuWrapper::graph_neighbor_sample_all_edge_type(
    int gpu_id,
    int edge_type_len,
    uint64_t *key,
    int sample_size,
    int len,
    std::vector<std::shared_ptr<phi::Allocation>> edge_type_graphs) {
  return ((GpuPsGraphTable *)graph_table)
      ->graph_neighbor_sample_all_edge_type(
          gpu_id, edge_type_len, key, sample_size, len, edge_type_graphs);
}

std::vector<std::shared_ptr<phi::Allocation>>
GraphGpuWrapper::get_edge_type_graph(int gpu_id, int edge_type_len) {
  return ((GpuPsGraphTable *)graph_table)
      ->get_edge_type_graph(gpu_id, edge_type_len);
}

int GraphGpuWrapper::get_feature_info_of_nodes(
    int gpu_id,
    uint64_t *d_nodes,
    int node_num,
    uint32_t *size_list,
    uint32_t *size_list_prefix_sum,
    std::shared_ptr<phi::Allocation> &feature_list,
    std::shared_ptr<phi::Allocation> &slot_list) {
  platform::CUDADeviceGuard guard(gpu_id);
  PADDLE_ENFORCE_NOT_NULL(graph_table,
                          paddle::platform::errors::InvalidArgument(
                              "graph_table should not be null"));
  return ((GpuPsGraphTable *)graph_table)
      ->get_feature_info_of_nodes(gpu_id,
                                  d_nodes,
                                  node_num,
                                  size_list,
                                  size_list_prefix_sum,
                                  feature_list,
                                  slot_list);
}

int GraphGpuWrapper::get_feature_of_nodes(int gpu_id,
                                          uint64_t *d_walk,
                                          uint64_t *d_offset,
                                          uint32_t size,
                                          int slot_num,
                                          int *d_slot_feature_num_map,
                                          int fea_num_per_node) {
  platform::CUDADeviceGuard guard(gpu_id);
  PADDLE_ENFORCE_NOT_NULL(graph_table,
                          paddle::platform::errors::InvalidArgument(
                              "graph_table should not be null"));
  return ((GpuPsGraphTable *)graph_table)
      ->get_feature_of_nodes(gpu_id,
                             d_walk,
                             d_offset,
                             size,
                             slot_num,
                             d_slot_feature_num_map,
                             fea_num_per_node);
}

NeighborSampleResult GraphGpuWrapper::graph_neighbor_sample(
    int gpu_id, uint64_t *device_keys, int walk_degree, int len) {
  platform::CUDADeviceGuard guard(gpu_id);
  auto neighbor_sample_res =
      ((GpuPsGraphTable *)graph_table)
          ->graph_neighbor_sample(gpu_id, device_keys, walk_degree, len);

  return neighbor_sample_res;
}

int GraphGpuWrapper::get_feature_of_nodes(int gpu_id,
                                          uint64_t *d_walk,
                                          uint64_t *d_offset,
                                          uint32_t size,
                                          int slot_num) {
  platform::CUDADeviceGuard guard(gpu_id);
  PADDLE_ENFORCE_NOT_NULL(graph_table,
                          paddle::platform::errors::InvalidArgument(
                              "graph_table should not be null"));
  return ((GpuPsGraphTable *)graph_table)
      ->get_feature_of_nodes(gpu_id, d_walk, d_offset, size, slot_num);
}

NeighborSampleResult GraphGpuWrapper::graph_neighbor_sample(
    int gpu_id, uint64_t *device_keys, int walk_degree, int len) {
  platform::CUDADeviceGuard guard(gpu_id);
  auto neighbor_sample_res =
      ((GpuPsGraphTable *)graph_table)
          ->graph_neighbor_sample(gpu_id, device_keys, walk_degree, len);

  return neighbor_sample_res;
}

// this function is contributed by Liwb5
std::vector<uint64_t> GraphGpuWrapper::graph_neighbor_sample(
    int gpu_id, int idx, std::vector<uint64_t> &key, int sample_size) {
  std::vector<uint64_t> res;
  if (key.size() == 0) {
    return res;
  }
  uint64_t *cuda_key;
  platform::CUDADeviceGuard guard(gpu_id);

  hipMalloc(&cuda_key, key.size() * sizeof(uint64_t));
  hipMemcpy(cuda_key,
             key.data(),
             key.size() * sizeof(uint64_t),
             hipMemcpyHostToDevice);
  VLOG(0) << "key_size: " << key.size();
  auto neighbor_sample_res =
      ((GpuPsGraphTable *)graph_table)
          ->graph_neighbor_sample_v2(
<<<<<<< HEAD
              gpu_id, idx, cuda_key, sample_size, key.size(), false);
=======
              gpu_id, idx, cuda_key, sample_size, key.size(), false, true);
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
  int *actual_sample_size = new int[key.size()];
  hipMemcpy(actual_sample_size,
             neighbor_sample_res.actual_sample_size,
             key.size() * sizeof(int),
             hipMemcpyDeviceToHost);  // 3, 1, 3
  int cumsum = 0;
  for (int i = 0; i < key.size(); i++) {
    cumsum += actual_sample_size[i];
  }

  std::vector<uint64_t> cpu_key;
  cpu_key.resize(key.size() * sample_size);

  hipMemcpy(cpu_key.data(),
             neighbor_sample_res.val,
             key.size() * sample_size * sizeof(uint64_t),
             hipMemcpyDeviceToHost);
  for (int i = 0; i < key.size(); i++) {
    for (int j = 0; j < actual_sample_size[i]; j++) {
      res.push_back(key[i]);
      res.push_back(cpu_key[i * sample_size + j]);
    }
  }
  delete[] actual_sample_size;
  hipFree(cuda_key);
  return res;
}

NodeQueryResult GraphGpuWrapper::query_node_list(int gpu_id,
                                                 int idx,
                                                 int start,
                                                 int query_size) {
  PADDLE_ENFORCE_EQ(FLAGS_gpugraph_load_node_list_into_hbm,
                    true,
                    paddle::platform::errors::PreconditionNotMet(
                        "when use query_node_list should set "
                        "gpugraph_load_node_list_into_hbm true"));
  return ((GpuPsGraphTable *)graph_table)
      ->query_node_list(gpu_id, idx, start, query_size);
}
void GraphGpuWrapper::load_node_weight(int type_id, int idx, std::string path) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->load_node_weight(type_id, idx, path);
<<<<<<< HEAD
=======
}

std::vector<int> GraphGpuWrapper::slot_feature_num_map() const {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->slot_feature_num_map();
>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
}

void GraphGpuWrapper::export_partition_files(int idx, std::string file_path) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->export_partition_files(idx, file_path);
}
<<<<<<< HEAD
=======

void GraphGpuWrapper::release_graph() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->release_graph();
}

void GraphGpuWrapper::release_graph_edge() {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->release_graph_edge();
}

void GraphGpuWrapper::release_graph_node() {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->release_graph_node();
}

std::vector<uint64_t> &GraphGpuWrapper::get_graph_total_keys() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->graph_total_keys_;
}

std::vector<std::vector<uint64_t>> &GraphGpuWrapper::get_graph_type_keys() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->graph_type_keys_;
}

std::unordered_map<int, int> &GraphGpuWrapper::get_graph_type_to_index() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->type_to_index_;
}

std::string &GraphGpuWrapper::get_node_type_size(std::string first_node_type) {
  auto node_types =
      paddle::string::split_string<std::string>(first_node_type, ";");
  for (auto &type : node_types) {
    uniq_first_node_.insert(type);
  }

  auto &graph_all_type_total_keys = get_graph_type_keys();
  auto &type_to_index = get_graph_type_to_index();
  std::vector<std::string> node_type_size;
  for (auto node : uniq_first_node_) {
    auto it = feature_to_id.find(node);
    auto first_node_idx = it->second;
    size_t f_idx = type_to_index[first_node_idx];
    int type_total_key_size = graph_all_type_total_keys[f_idx].size();
    std::string node_type_str =
        node + ":" + std::to_string(type_total_key_size);
    node_type_size.push_back(node_type_str);
  }
  std::string delim = ";";
  node_type_size_str_ = paddle::string::join_strings(node_type_size, delim);

  return node_type_size_str_;
}

std::string &GraphGpuWrapper::get_edge_type_size() {
  auto edge_type_size =
      ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->edge_type_size;
  std::string delim = ";";
  edge_type_size_str_ = paddle::string::join_strings(edge_type_size, delim);
  std::cout << "edge_type_size_str: " << edge_type_size_str_ << std::endl;
  return edge_type_size_str_;
}

>>>>>>> e170b253fc2cfc81aeb39c17a0fffc8e08311f1e
#endif
}  // namespace framework
};  // namespace paddle
