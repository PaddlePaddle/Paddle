// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_wrapper.h"
#include <sstream>
#include "paddle/fluid/framework/fleet/heter_ps/gpu_graph_utils.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_ps_table.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_resource.h"
DECLARE_int32(gpugraph_storage_mode);
namespace paddle {
namespace framework {
#ifdef PADDLE_WITH_HETERPS

std::shared_ptr<GraphGpuWrapper> GraphGpuWrapper::s_instance_(nullptr);
void GraphGpuWrapper::set_device(std::vector<int> ids) {
  for (auto device_id : ids) {
    device_id_mapping.push_back(device_id);
  }
}

<<<<<<< HEAD
void GraphGpuWrapper::init_conf(const std::string &first_node_type,
                                const std::string &meta_path) {
  static std::mutex mutex;
  {
    std::lock_guard<std::mutex> lock(mutex);
    if (conf_initialized_) {
      return;
    }
    VLOG(2) << "init path config";
    conf_initialized_ = true;
    auto node_types =
        paddle::string::split_string<std::string>(first_node_type, ";");
    VLOG(2) << "node_types: " << first_node_type;
    for (auto &type : node_types) {
      auto iter = feature_to_id.find(type);
      PADDLE_ENFORCE_NE(iter,
                        feature_to_id.end(),
                        platform::errors::NotFound(
                            "(%s) is not found in feature_to_id.", type));
      VLOG(2) << "feature_to_id[" << type << "] = " << iter->second;
      first_node_type_.push_back(iter->second);
    }
    meta_path_.resize(first_node_type_.size());
    auto meta_paths = paddle::string::split_string<std::string>(meta_path, ";");

    for (size_t i = 0; i < meta_paths.size(); i++) {
      auto path = meta_paths[i];
      auto nodes = paddle::string::split_string<std::string>(path, "-");
      for (auto &node : nodes) {
        auto iter = edge_to_id.find(node);
        PADDLE_ENFORCE_NE(iter,
                          edge_to_id.end(),
                          platform::errors::NotFound(
                              "(%s) is not found in edge_to_id.", node));
        VLOG(2) << "edge_to_id[" << node << "] = " << iter->second;
        meta_path_[i].push_back(iter->second);
      }
    }
    int max_dev_id = 0;
    for (size_t i = 0; i < device_id_mapping.size(); i++) {
      if (device_id_mapping[i] > max_dev_id) {
        max_dev_id = device_id_mapping[i];
      }
    }
    finish_node_type_.resize(max_dev_id + 1);
    node_type_start_.resize(max_dev_id + 1);
    global_infer_node_type_start_.resize(max_dev_id + 1);
    for (size_t i = 0; i < device_id_mapping.size(); i++) {
      int dev_id = device_id_mapping[i];
      auto &node_type_start = node_type_start_[i];
      auto &infer_node_type_start = global_infer_node_type_start_[i];
      auto &finish_node_type = finish_node_type_[i];
      finish_node_type.clear();

      for (size_t idx = 0; idx < feature_to_id.size(); idx++) {
        infer_node_type_start[idx] = 0;
      }
      for (auto &type : node_types) {
        auto iter = feature_to_id.find(type);
        node_type_start[iter->second] = 0;
        infer_node_type_start[iter->second] = 0;
      }
      infer_cursor_.push_back(0);
      cursor_.push_back(0);
    }
    init_type_keys();
  }
}

void GraphGpuWrapper::init_type_keys() {
  size_t thread_num = device_id_mapping.size();
  int cnt = 0;

  auto &graph_all_type_total_keys = get_graph_type_keys();
  auto &type_to_index = get_graph_type_to_index();
  std::vector<std::vector<uint64_t>> tmp_keys;
  tmp_keys.resize(thread_num);
  d_graph_all_type_total_keys_.resize(graph_all_type_total_keys.size());
  h_graph_all_type_keys_len_.resize(graph_all_type_total_keys.size());
  for (size_t f_idx = 0; f_idx < graph_all_type_total_keys.size(); f_idx++) {
    for (size_t j = 0; j < tmp_keys.size(); j++) {
      tmp_keys[j].clear();
    }
    d_graph_all_type_total_keys_[f_idx].resize(thread_num);
    auto &type_total_key = graph_all_type_total_keys[f_idx];
    for (size_t j = 0; j < type_total_key.size(); j++) {
      uint64_t shard = type_total_key[j] % thread_num;
      tmp_keys[shard].push_back(type_total_key[j]);
    }
    for (size_t j = 0; j < thread_num; j++) {
      h_graph_all_type_keys_len_[f_idx].push_back(tmp_keys[j].size());
      VLOG(1) << "node type: " << type_to_index[f_idx]
              << ", gpu_graph_device_keys[" << j
              << "] = " << tmp_keys[j].size();
    }
    for (size_t j = 0; j < thread_num; j++) {
      auto stream = get_local_stream(j);
      int gpuid = device_id_mapping[j];
      auto place = platform::CUDAPlace(gpuid);
      platform::CUDADeviceGuard guard(gpuid);
      d_graph_all_type_total_keys_[f_idx][j] =
          memory::AllocShared(place, tmp_keys[j].size() * sizeof(uint64_t));
      hipMemcpyAsync(d_graph_all_type_total_keys_[f_idx][j]->ptr(),
                      tmp_keys[j].data(),
                      sizeof(uint64_t) * tmp_keys[j].size(),
                      hipMemcpyHostToDevice,
                      stream);
    }
  }
  for (int i = 0; i < thread_num; i++) {
    auto stream = get_local_stream(i);
    hipStreamSynchronize(stream);
  }
}

=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
int GraphGpuWrapper::get_all_id(int type,
                                int slice_num,
                                std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, slice_num, output);
}

int GraphGpuWrapper::get_all_neighbor_id(
    int type, int slice_num, std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_neighbor_id(type, slice_num, output);
}

int GraphGpuWrapper::get_all_id(int type,
                                int idx,
                                int slice_num,
                                std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, idx, slice_num, output);
}

int GraphGpuWrapper::get_all_neighbor_id(
    int type,
    int idx,
    int slice_num,
    std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_neighbor_id(type, idx, slice_num, output);
}

int GraphGpuWrapper::get_all_feature_ids(
    int type,
    int idx,
    int slice_num,
    std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_feature_ids(type, idx, slice_num, output);
}

<<<<<<< HEAD
int GraphGpuWrapper::get_node_embedding_ids(
    int slice_num, std::vector<std::vector<uint64_t>> *output) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_node_embedding_ids(slice_num, output);
}

=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
void GraphGpuWrapper::set_up_types(std::vector<std::string> &edge_types,
                                   std::vector<std::string> &node_types) {
  id_to_edge = edge_types;
  for (size_t table_id = 0; table_id < edge_types.size(); table_id++) {
    int res = edge_to_id.size();
    edge_to_id[edge_types[table_id]] = res;
  }
  id_to_feature = node_types;
  for (size_t table_id = 0; table_id < node_types.size(); table_id++) {
    int res = feature_to_id.size();
    feature_to_id[node_types[table_id]] = res;
  }
  table_feat_mapping.resize(node_types.size());
  this->table_feat_conf_feat_name.resize(node_types.size());
  this->table_feat_conf_feat_dtype.resize(node_types.size());
  this->table_feat_conf_feat_shape.resize(node_types.size());
}

void GraphGpuWrapper::set_feature_separator(std::string ch) {
  feature_separator_ = ch;
  if (graph_table != nullptr) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->set_feature_separator(feature_separator_);
  }
}

<<<<<<< HEAD
void GraphGpuWrapper::set_slot_feature_separator(std::string ch) {
  slot_feature_separator_ = ch;
  if (graph_table != nullptr) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->set_slot_feature_separator(slot_feature_separator_);
  }
}

=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
void GraphGpuWrapper::make_partitions(int idx,
                                      int64_t byte_size,
                                      int device_len) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->make_partitions(idx, byte_size, device_len);
}
int32_t GraphGpuWrapper::load_next_partition(int idx) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->load_next_partition(idx);
}

void GraphGpuWrapper::set_search_level(int level) {
  ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->set_search_level(level);
}

std::vector<uint64_t> GraphGpuWrapper::get_partition(int idx, int num) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_partition(idx, num);
}
int32_t GraphGpuWrapper::get_partition_num(int idx) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_partition_num(idx);
}
void GraphGpuWrapper::make_complementary_graph(int idx, int64_t byte_size) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->make_complementary_graph(idx, byte_size);
}
void GraphGpuWrapper::load_edge_file(std::string name,
                                     std::string filepath,
                                     bool reverse) {
  // 'e' means load edge
  std::string params = "e";
  if (reverse) {
    // 'e<' means load edges from $2 to $1
    params += "<" + name;
  } else {
    // 'e>' means load edges from $1 to $2
    params += ">" + name;
  }
  if (edge_to_id.find(name) != edge_to_id.end()) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->Load(std::string(filepath), params);
  }
}

void GraphGpuWrapper::load_edge_file(std::string etype2files,
                                     std::string graph_data_local_path,
                                     int part_num,
                                     bool reverse) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->parse_edge_and_load(
          etype2files, graph_data_local_path, part_num, reverse);
}

void GraphGpuWrapper::load_node_file(std::string name, std::string filepath) {
  // 'n' means load nodes and 'node_type' follows

  std::string params = "n" + name;

  if (feature_to_id.find(name) != feature_to_id.end()) {
    ((GpuPsGraphTable *)graph_table)
        ->cpu_graph_table_->Load(std::string(filepath), params);
  }
}

<<<<<<< HEAD
void GraphGpuWrapper::load_node_file(std::string ntype2files,
                                     std::string graph_data_local_path,
                                     int part_num) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->parse_node_and_load(
          ntype2files, graph_data_local_path, part_num);
}

void GraphGpuWrapper::load_node_and_edge(std::string etype2files,
                                         std::string ntype2files,
                                         std::string graph_data_local_path,
=======
void GraphGpuWrapper::load_node_and_edge(std::string etype,
                                         std::string ntype,
                                         std::string epath,
                                         std::string npath,
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
                                         int part_num,
                                         bool reverse) {
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->load_node_and_edge_file(
<<<<<<< HEAD
          etype2files, ntype2files, graph_data_local_path, part_num, reverse);
=======
          etype, ntype, epath, npath, part_num, reverse);
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
}

void GraphGpuWrapper::add_table_feat_conf(std::string table_name,
                                          std::string feat_name,
                                          std::string feat_dtype,
                                          int feat_shape) {
  if (feature_to_id.find(table_name) != feature_to_id.end()) {
    int idx = feature_to_id[table_name];
    if (table_feat_mapping[idx].find(feat_name) ==
        table_feat_mapping[idx].end()) {
      int res = (int)table_feat_mapping[idx].size();
      table_feat_mapping[idx][feat_name] = res;
    }
    int feat_idx = table_feat_mapping[idx][feat_name];
    VLOG(0) << "table_name " << table_name << " mapping id " << idx;
    VLOG(0) << " feat name " << feat_name << " feat id" << feat_idx;
    if (feat_idx < table_feat_conf_feat_name[idx].size()) {
      // overide
      table_feat_conf_feat_name[idx][feat_idx] = feat_name;
      table_feat_conf_feat_dtype[idx][feat_idx] = feat_dtype;
      table_feat_conf_feat_shape[idx][feat_idx] = feat_shape;
    } else {
      // new
      table_feat_conf_feat_name[idx].push_back(feat_name);
      table_feat_conf_feat_dtype[idx].push_back(feat_dtype);
      table_feat_conf_feat_shape[idx].push_back(feat_shape);
    }
  }
  VLOG(0) << "add conf over";
}
void GraphGpuWrapper::init_search_level(int level) { search_level = level; }

gpuStream_t GraphGpuWrapper::get_local_stream(int gpuid) {
  return ((GpuPsGraphTable *)graph_table)->get_local_stream(gpuid);
}

void GraphGpuWrapper::init_service() {
<<<<<<< HEAD
  table_proto.set_task_pool_size(64);
=======
  table_proto.set_task_pool_size(24);
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
  table_proto.set_shard_num(1000);
  table_proto.set_build_sampler_on_cpu(false);
  table_proto.set_search_level(search_level);
  table_proto.set_table_name("cpu_graph_table_");
  table_proto.set_use_cache(false);
  for (int i = 0; i < id_to_edge.size(); i++)
    table_proto.add_edge_types(id_to_edge[i]);
  for (int i = 0; i < id_to_feature.size(); i++) {
    table_proto.add_node_types(id_to_feature[i]);
    auto feat_node = id_to_feature[i];
    ::paddle::distributed::GraphFeature *g_f = table_proto.add_graph_feature();
    for (int x = 0; x < table_feat_conf_feat_name[i].size(); x++) {
      g_f->add_name(table_feat_conf_feat_name[i][x]);
      g_f->add_dtype(table_feat_conf_feat_dtype[i][x]);
      g_f->add_shape(table_feat_conf_feat_shape[i][x]);
    }
  }
  std::shared_ptr<HeterPsResource> resource =
      std::make_shared<HeterPsResource>(device_id_mapping);
  resource->enable_p2p();
<<<<<<< HEAD
  GpuPsGraphTable *g = new GpuPsGraphTable(resource, id_to_edge.size());
  size_t gpu_num = device_id_mapping.size();
  g->init_cpu_table(table_proto, gpu_num);
  g->cpu_graph_table_->set_feature_separator(feature_separator_);
  g->cpu_graph_table_->set_slot_feature_separator(slot_feature_separator_);
  graph_table = (char *)g;
  upload_num = gpu_num;
=======
  GpuPsGraphTable *g = new GpuPsGraphTable(resource, 1, id_to_edge.size());
  g->init_cpu_table(table_proto);
  g->cpu_graph_table_->set_feature_separator(feature_separator_);
  graph_table = (char *)g;
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
  upload_task_pool.reset(new ::ThreadPool(upload_num));
}

void GraphGpuWrapper::finalize() {
  ((GpuPsGraphTable *)graph_table)->show_table_collisions();
}

void GraphGpuWrapper::upload_batch(int type,
                                   int idx,
                                   int slice_num,
                                   const std::string &edge_type) {
  VLOG(0) << "begin upload edge, type[" << edge_type << "]";
  std::vector<std::vector<uint64_t>> ids;
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, idx, slice_num, &ids);
  debug_gpu_memory_info("upload_batch node start");
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;

  for (int i = 0; i < ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, idx, this]() -> int {
      VLOG(0) << "begin make_gpu_ps_graph, node_id[" << i << "]_size["
              << ids[i].size() << "]";
      GpuPsCommGraph sub_graph =
          g->cpu_graph_table_->make_gpu_ps_graph(idx, ids[i]);
      g->build_graph_on_single_gpu(sub_graph, i, idx);
      sub_graph.release_on_cpu();
      VLOG(0) << "sub graph on gpu " << i << " is built";
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  debug_gpu_memory_info("upload_batch node end");
}

// feature table
void GraphGpuWrapper::upload_batch(int type, int slice_num, int slot_num) {
<<<<<<< HEAD
  if (type == 1 &&
      (FLAGS_gpugraph_storage_mode == paddle::framework::GpuGraphStorageMode::
                                          MEM_EMB_FEATURE_AND_GPU_GRAPH ||
       FLAGS_gpugraph_storage_mode == paddle::framework::GpuGraphStorageMode::
                                          SSD_EMB_AND_MEM_FEATURE_GPU_GRAPH)) {
    return;
  }
=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
  std::vector<std::vector<uint64_t>> node_ids;
  ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->get_all_id(type, slice_num, &node_ids);
  debug_gpu_memory_info("upload_batch feature start");
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;
  for (int i = 0; i < node_ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, this]() -> int {
      VLOG(0) << "begin make_gpu_ps_graph_fea, node_ids[" << i << "]_size["
              << node_ids[i].size() << "]";
      GpuPsCommGraphFea sub_graph =
<<<<<<< HEAD
          g->cpu_graph_table_->make_gpu_ps_graph_fea(i, node_ids[i], slot_num);
=======
          g->cpu_graph_table_->make_gpu_ps_graph_fea(node_ids[i], slot_num);
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
      // sub_graph.display_on_cpu();
      VLOG(0) << "begin build_graph_fea_on_single_gpu, node_ids[" << i
              << "]_size[" << node_ids[i].size() << "]";
      g->build_graph_fea_on_single_gpu(sub_graph, i);
      sub_graph.release_on_cpu();
      VLOG(0) << "sub graph fea on gpu " << i << " is built";
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  // g->build_graph_from_cpu(vec);
  debug_gpu_memory_info("upload_batch feature end");
<<<<<<< HEAD
}

// get sub_graph_fea
std::vector<GpuPsCommGraphFea> GraphGpuWrapper::get_sub_graph_fea(
    std::vector<std::vector<uint64_t>> &node_ids, int slot_num) {
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  std::vector<std::future<int>> tasks;
  std::vector<GpuPsCommGraphFea> sub_graph_feas(node_ids.size());
  for (int i = 0; i < node_ids.size(); i++) {
    tasks.push_back(upload_task_pool->enqueue([&, i, this]() -> int {
      GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
      sub_graph_feas[i] =
          g->cpu_graph_table_->make_gpu_ps_graph_fea(i, node_ids[i], slot_num);
      return 0;
    }));
  }
  for (size_t i = 0; i < tasks.size(); i++) tasks[i].get();
  return sub_graph_feas;
}

// build_gpu_graph_fea
void GraphGpuWrapper::build_gpu_graph_fea(GpuPsCommGraphFea &sub_graph_fea,
                                          int i) {
  GpuPsGraphTable *g = (GpuPsGraphTable *)graph_table;
  g->build_graph_fea_on_single_gpu(sub_graph_fea, i);
  sub_graph_fea.release_on_cpu();
  VLOG(0) << "sub graph fea on gpu " << i << " is built";
  return;
=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
}

NeighborSampleResult GraphGpuWrapper::graph_neighbor_sample_v3(
    NeighborSampleQuery q, bool cpu_switch, bool compress = true) {
  return ((GpuPsGraphTable *)graph_table)
      ->graph_neighbor_sample_v3(q, cpu_switch, compress);
}

NeighborSampleResultV2 GraphGpuWrapper::graph_neighbor_sample_all_edge_type(
    int gpu_id,
    int edge_type_len,
    uint64_t *key,
    int sample_size,
    int len,
    std::vector<std::shared_ptr<phi::Allocation>> edge_type_graphs) {
  return ((GpuPsGraphTable *)graph_table)
      ->graph_neighbor_sample_all_edge_type(
          gpu_id, edge_type_len, key, sample_size, len, edge_type_graphs);
}

std::vector<std::shared_ptr<phi::Allocation>>
GraphGpuWrapper::get_edge_type_graph(int gpu_id, int edge_type_len) {
  return ((GpuPsGraphTable *)graph_table)
      ->get_edge_type_graph(gpu_id, edge_type_len);
}

int GraphGpuWrapper::get_feature_info_of_nodes(
    int gpu_id,
    uint64_t *d_nodes,
    int node_num,
    uint32_t *size_list,
    uint32_t *size_list_prefix_sum,
    std::shared_ptr<phi::Allocation> &feature_list,
    std::shared_ptr<phi::Allocation> &slot_list) {
  platform::CUDADeviceGuard guard(gpu_id);
  PADDLE_ENFORCE_NOT_NULL(graph_table,
                          paddle::platform::errors::InvalidArgument(
                              "graph_table should not be null"));
  return ((GpuPsGraphTable *)graph_table)
      ->get_feature_info_of_nodes(gpu_id,
                                  d_nodes,
                                  node_num,
                                  size_list,
                                  size_list_prefix_sum,
                                  feature_list,
                                  slot_list);
}

int GraphGpuWrapper::get_feature_of_nodes(int gpu_id,
                                          uint64_t *d_walk,
                                          uint64_t *d_offset,
                                          uint32_t size,
                                          int slot_num,
                                          int *d_slot_feature_num_map,
                                          int fea_num_per_node) {
  platform::CUDADeviceGuard guard(gpu_id);
  PADDLE_ENFORCE_NOT_NULL(graph_table,
                          paddle::platform::errors::InvalidArgument(
                              "graph_table should not be null"));
  return ((GpuPsGraphTable *)graph_table)
      ->get_feature_of_nodes(gpu_id,
                             d_walk,
                             d_offset,
                             size,
                             slot_num,
                             d_slot_feature_num_map,
                             fea_num_per_node);
}

NeighborSampleResult GraphGpuWrapper::graph_neighbor_sample(
    int gpu_id, uint64_t *device_keys, int walk_degree, int len) {
  platform::CUDADeviceGuard guard(gpu_id);
  auto neighbor_sample_res =
      ((GpuPsGraphTable *)graph_table)
          ->graph_neighbor_sample(gpu_id, device_keys, walk_degree, len);

  return neighbor_sample_res;
}

int GraphGpuWrapper::get_feature_of_nodes(int gpu_id,
                                          uint64_t *d_walk,
                                          uint64_t *d_offset,
                                          uint32_t size,
                                          int slot_num) {
  platform::CUDADeviceGuard guard(gpu_id);
  PADDLE_ENFORCE_NOT_NULL(graph_table,
                          paddle::platform::errors::InvalidArgument(
                              "graph_table should not be null"));
  return ((GpuPsGraphTable *)graph_table)
      ->get_feature_of_nodes(gpu_id, d_walk, d_offset, size, slot_num);
}

NeighborSampleResult GraphGpuWrapper::graph_neighbor_sample(
    int gpu_id, uint64_t *device_keys, int walk_degree, int len) {
  platform::CUDADeviceGuard guard(gpu_id);
  auto neighbor_sample_res =
      ((GpuPsGraphTable *)graph_table)
          ->graph_neighbor_sample(gpu_id, device_keys, walk_degree, len);

  return neighbor_sample_res;
}

// this function is contributed by Liwb5
std::vector<uint64_t> GraphGpuWrapper::graph_neighbor_sample(
    int gpu_id, int idx, std::vector<uint64_t> &key, int sample_size) {
  std::vector<uint64_t> res;
  if (key.size() == 0) {
    return res;
  }
  uint64_t *cuda_key;
  platform::CUDADeviceGuard guard(gpu_id);

  hipMalloc(&cuda_key, key.size() * sizeof(uint64_t));
  hipMemcpy(cuda_key,
             key.data(),
             key.size() * sizeof(uint64_t),
             hipMemcpyHostToDevice);
  VLOG(0) << "key_size: " << key.size();
  auto neighbor_sample_res =
      ((GpuPsGraphTable *)graph_table)
          ->graph_neighbor_sample_v2(
<<<<<<< HEAD
              gpu_id, idx, cuda_key, sample_size, key.size(), false, true);
=======
              gpu_id, idx, cuda_key, sample_size, key.size(), false);
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
  int *actual_sample_size = new int[key.size()];
  hipMemcpy(actual_sample_size,
             neighbor_sample_res.actual_sample_size,
             key.size() * sizeof(int),
             hipMemcpyDeviceToHost);  // 3, 1, 3
  int cumsum = 0;
  for (int i = 0; i < key.size(); i++) {
    cumsum += actual_sample_size[i];
  }

  std::vector<uint64_t> cpu_key;
  cpu_key.resize(key.size() * sample_size);

  hipMemcpy(cpu_key.data(),
             neighbor_sample_res.val,
             key.size() * sample_size * sizeof(uint64_t),
             hipMemcpyDeviceToHost);
  for (int i = 0; i < key.size(); i++) {
    for (int j = 0; j < actual_sample_size[i]; j++) {
      res.push_back(key[i]);
      res.push_back(cpu_key[i * sample_size + j]);
    }
  }
  delete[] actual_sample_size;
  hipFree(cuda_key);
  return res;
}

NodeQueryResult GraphGpuWrapper::query_node_list(int gpu_id,
                                                 int idx,
                                                 int start,
                                                 int query_size) {
  PADDLE_ENFORCE_EQ(FLAGS_gpugraph_load_node_list_into_hbm,
                    true,
                    paddle::platform::errors::PreconditionNotMet(
                        "when use query_node_list should set "
                        "gpugraph_load_node_list_into_hbm true"));
  return ((GpuPsGraphTable *)graph_table)
      ->query_node_list(gpu_id, idx, start, query_size);
}
void GraphGpuWrapper::load_node_weight(int type_id, int idx, std::string path) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->load_node_weight(type_id, idx, path);
<<<<<<< HEAD
}

std::vector<int> GraphGpuWrapper::slot_feature_num_map() const {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->slot_feature_num_map();
=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
}

void GraphGpuWrapper::export_partition_files(int idx, std::string file_path) {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->export_partition_files(idx, file_path);
}
<<<<<<< HEAD

void GraphGpuWrapper::release_graph() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->release_graph();
}

void GraphGpuWrapper::release_graph_edge() {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->release_graph_edge();
}

void GraphGpuWrapper::release_graph_node() {
  return ((GpuPsGraphTable *)graph_table)
      ->cpu_graph_table_->release_graph_node();
}

std::vector<uint64_t> &GraphGpuWrapper::get_graph_total_keys() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->graph_total_keys_;
}

std::vector<std::vector<uint64_t>> &GraphGpuWrapper::get_graph_type_keys() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->graph_type_keys_;
}

std::unordered_map<int, int> &GraphGpuWrapper::get_graph_type_to_index() {
  return ((GpuPsGraphTable *)graph_table)->cpu_graph_table_->type_to_index_;
}

=======
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
#endif
}  // namespace framework
};  // namespace paddle
