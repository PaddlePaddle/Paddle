#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/tensor_util.h"

#include <algorithm>
#include <limits>
#include <memory>
#include <string>
#include <utility>
#include <vector>

#include "paddle/fluid/framework/data_type.h"
#include "paddle/fluid/platform/complex128.h"
#include "paddle/fluid/platform/complex64.h"
#include "paddle/fluid/platform/profiler.h"

namespace paddle {
namespace framework {

void TensorCopy(const Tensor& src, const platform::Place& dst_place,
                const platform::DeviceContext& ctx, Tensor* dst) {
  if (&src == dst) {
    auto src_copy = src;
    TensorCopy(src_copy, dst_place, ctx, dst);
    return;
  }

  VLOG(3) << "TensorCopy " << src.dims() << " from " << src.place() << " to "
          << dst_place;
  src.check_memory_size();

  dst->Resize(src.dims());
  dst->set_layout(src.layout());
  auto src_place = src.place();
  auto src_ptr = src.data<void>();
#ifdef PADDLE_WITH_MKLDNN
  dst->set_format(src.format());
  // oneDNN tensors due to padding may be of bigger size
  // than numel()*size(type())
  auto dst_ptr =
      src.layout() == DataLayout::kMKLDNN
          ? dst->mutable_data(dst_place, src.type(), src.memory_size())
          : dst->mutable_data(dst_place, src.type());
#else
  auto dst_ptr = dst->mutable_data(dst_place, src.type());
#endif
  if (src_ptr == dst_ptr && src_place == dst_place) {
    VLOG(3) << "Skip copy the same data async from " << src_place << " to "
            << dst_place;
    return;
  }

#ifdef PADDLE_WITH_MKLDNN
  auto size = src.layout() == DataLayout::kMKLDNN
                  ? src.memory_size()
                  : src.numel() * SizeOfType(src.type());
#else
  auto size = src.numel() * SizeOfType(src.type());
#endif

  if (platform::is_cpu_place(src_place) && platform::is_cpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CPUPlace, src_place), src_ptr, size);
  }
#ifdef PADDLE_WITH_XPU
  else if (platform::is_xpu_place(src_place) &&  // NOLINT
           platform::is_cpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::XPUPlace, src_place), src_ptr, size);
  } else if (platform::is_cpu_place(src_place) &&
             platform::is_xpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::XPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CPUPlace, src_place), src_ptr, size);
  } else if (platform::is_xpu_place(src_place) &&
             platform::is_xpu_place(dst_place)) {
    if (src_ptr == dst_ptr) {
      VLOG(3) << "Skip copy the same data async from " << src_place << " to "
              << dst_place;
      return;
    }
    memory::Copy(BOOST_GET_CONST(platform::XPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::XPUPlace, src_place), src_ptr, size);
  } else {
    PADDLE_THROW(platform::errors::Unimplemented(
        "Copy from %s to %s is not supported.", src_place, dst_place));
  }
#endif
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
  else if (platform::is_cuda_pinned_place(src_place) &&  // NOLINT
           platform::is_cuda_pinned_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CUDAPinnedPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CUDAPinnedPlace, src_place), src_ptr,
                 size);
  }
  else if (platform::is_cuda_pinned_place(src_place) &&  // NOLINT
           platform::is_cpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CUDAPinnedPlace, src_place), src_ptr,
                 size);
  }
  else if (platform::is_cpu_place(src_place) &&  // NOLINT
           platform::is_cuda_pinned_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CUDAPinnedPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CPUPlace, src_place), src_ptr, size);
  }
  else if (platform::is_gpu_place(src_place) &&  // NOLINT
           platform::is_cpu_place(dst_place)) {
    auto src_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, src_place);
    auto dst_cpu_place = BOOST_GET_CONST(platform::CPUPlace, dst_place);
    auto ctx_place = ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx_place), true,
        platform::errors::PreconditionNotMet(
            "Context place error, excepted GPUPlace, but actually %s.",
            ctx_place));
    auto ctx_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(src_gpu_place, ctx_gpu_place,
                      platform::errors::Unavailable(
                          "Source place and context place do not match, source "
                          "place is %s, context place is %s.",
                          src_gpu_place, ctx_gpu_place));
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    memory::Copy(dst_cpu_place, dst_ptr, src_gpu_place, src_ptr, size, stream);
  }
  else if (platform::is_cpu_place(src_place) &&  // NOLINT
           platform::is_gpu_place(dst_place)) {
    auto src_cpu_place = BOOST_GET_CONST(platform::CPUPlace, src_place);
    auto dst_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, dst_place);
    auto ctx_place = ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx_place), true,
        platform::errors::PreconditionNotMet(
            "Context place error, excepted GPUPlace, but actually %s.",
            ctx_place));
    auto ctx_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(dst_gpu_place, ctx_gpu_place,
                      platform::errors::Unavailable(
                          "Destination place and context place do not match, "
                          "destination place is %s, context place is %s.",
                          dst_gpu_place, ctx_gpu_place));
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    memory::Copy(dst_gpu_place, dst_ptr, src_cpu_place, src_ptr, size, stream);
  }
  else if (platform::is_gpu_place(src_place) &&  // NOLINT
           platform::is_cuda_pinned_place(dst_place)) {
    auto src_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, src_place);
    auto dst_cuda_pinned_place =
        BOOST_GET_CONST(platform::CUDAPinnedPlace, dst_place);
    auto ctx_place = ctx.GetPlace();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx_place), true,
                      platform::errors::PreconditionNotMet(
                          "Device context place mismatch. When copying Tensor "
                          "data from GPU memory to CUDA Pinned memory, current "
                          "device context place should be GPU."));
    auto ctx_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(src_gpu_place, ctx_gpu_place,
                      platform::errors::PreconditionNotMet(
                          "The source GPU device and current device context do "
                          "not match. The source GPU device number is %d, but "
                          "device context GPU number is %d.",
                          src_gpu_place.device, ctx_gpu_place.device));
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    memory::Copy(dst_cuda_pinned_place, dst_ptr, src_gpu_place, src_ptr, size,
                 stream);
  }
  else if (platform::is_cuda_pinned_place(src_place) &&  // NOLINT
           platform::is_gpu_place(dst_place)) {
    auto src_cuda_pinned_place =
        BOOST_GET_CONST(platform::CUDAPinnedPlace, src_place);
    auto dst_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, dst_place);
    auto ctx_place = ctx.GetPlace();
    PADDLE_ENFORCE_EQ(platform::is_gpu_place(ctx_place), true,
                      platform::errors::PreconditionNotMet(
                          "Device context place mismatch. When copying Tensor "
                          "data from CUDA Pinned memory to GPU memory, current "
                          "device context place should be GPU."));
    auto ctx_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, ctx_place);
    PADDLE_ENFORCE_EQ(dst_gpu_place, ctx_gpu_place,
                      platform::errors::PreconditionNotMet(
                          "The target GPU device and current device context do "
                          "not match. The target GPU device number is %d, but "
                          "device context GPU number is %d.",
                          dst_gpu_place.device, ctx_gpu_place.device));
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    memory::Copy(dst_gpu_place, dst_ptr, src_cuda_pinned_place, src_ptr, size,
                 stream);
  }
  else if (platform::is_gpu_place(src_place) &&  // NOLINT
           platform::is_gpu_place(dst_place)) {
    auto src_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, src_place);
    auto dst_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, dst_place);
    auto ctx_place = ctx.GetPlace();
    PADDLE_ENFORCE_EQ(
        platform::is_gpu_place(ctx_place), true,
        platform::errors::PreconditionNotMet(
            "Context place error, excepted GPUPlace, but actually %s.",
            ctx_place));
    auto stream =
        reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream();
    if (platform::is_same_place(src_place, dst_place)) {
      memory::Copy(dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size,
                   stream);
    } else {
      if (platform::is_same_place(ctx_place, src_place)) {
        memory::Copy(dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size,
                     stream);
        platform::DeviceContextPool::Instance().Get(src.place())->Wait();
      } else if (platform::is_same_place(ctx_place, dst_place)) {
        platform::DeviceContextPool::Instance().Get(src.place())->Wait();
        memory::Copy(dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size,
                     stream);
      } else {
        PADDLE_THROW(platform::errors::Unavailable(
            "Context place dose not match the source and destination place."));
      }
    }
  }
  else {  // NOLINT
    PADDLE_THROW(platform::errors::Unimplemented(
        "Copying from %s to %s is not supported.", src_place, dst_place));
  }
#endif
}

void TensorCopy(const Tensor& src, const platform::Place& dst_place,
                Tensor* dst) {
  platform::DeviceContextPool& pool = platform::DeviceContextPool::Instance();
  const platform::DeviceContext* dev_ctx;
  if (platform::is_gpu_place(dst_place)) {
    dev_ctx = pool.Get(dst_place);
  } else {
    dev_ctx = pool.Get(src.place());
  }
  TensorCopy(src, dst_place, *dev_ctx, dst);
}

void TensorCopySync(const Tensor& src, const platform::Place& dst_place,
                    Tensor* dst) {
  if (&src == dst) {
    auto src_copy = src;
    TensorCopySync(src_copy, dst_place, dst);
    return;
  }

  VLOG(3) << "TensorCopySync " << src.dims() << " from " << src.place()
          << " to " << dst_place;
  src.check_memory_size();
  dst->Resize(src.dims());
  dst->set_layout(src.layout());
#ifdef PADDLE_WITH_MKLDNN
  dst->set_format(src.format());
#endif
  auto src_place = src.place();
  auto src_ptr = src.data<void>();
  auto dst_ptr = dst->mutable_data(dst_place, src.type());

  if (src_ptr == dst_ptr && src_place == dst_place) {
    VLOG(3) << "Skip copy the same data from " << src_place << " to "
            << dst_place;
    return;
  }

  auto size = src.numel() * SizeOfType(src.type());
  if (platform::is_cpu_place(src_place) && platform::is_cpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CPUPlace, src_place), src_ptr, size);
  }
#ifdef PADDLE_WITH_XPU
  else if (platform::is_xpu_place(src_place) &&  // NOLINT
           platform::is_cpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::XPUPlace, src_place), src_ptr, size);
  } else if (platform::is_cpu_place(src_place) &&  // NOLINT
             platform::is_xpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::XPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CPUPlace, src_place), src_ptr, size);
  } else if (platform::is_xpu_place(src_place) &&  // NOLINT
             platform::is_xpu_place(dst_place)) {
    if (src_ptr == dst_ptr) {
      VLOG(3) << "Skip copy the same data async from " << src_place << " to "
              << dst_place;
      return;
    }
    memory::Copy(BOOST_GET_CONST(platform::XPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::XPUPlace, src_place), src_ptr, size);
  } else {  // NOLINT
    PADDLE_THROW(platform::errors::Unimplemented(
        "Copy from %s to %s is not supported.", src_place, dst_place));
  }
#endif
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
  else if (platform::is_cuda_pinned_place(src_place) &&  // NOLINT
           platform::is_cuda_pinned_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CUDAPinnedPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CUDAPinnedPlace, src_place), src_ptr,
                 size);
  }
  else if (platform::is_cuda_pinned_place(src_place) &&  // NOLINT
           platform::is_cpu_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CPUPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CUDAPinnedPlace, src_place), src_ptr,
                 size);
  }
  else if (platform::is_cpu_place(src_place) &&  // NOLINT
           platform::is_cuda_pinned_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CUDAPinnedPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CPUPlace, src_place), src_ptr, size);
  }
  else if (platform::is_gpu_place(src_place) &&  // NOLINT
           platform::is_cuda_pinned_place(dst_place)) {
    memory::Copy(BOOST_GET_CONST(platform::CUDAPinnedPlace, dst_place), dst_ptr,
                 BOOST_GET_CONST(platform::CUDAPlace, src_place), src_ptr, size,
                 nullptr);
  }
  else if (platform::is_gpu_place(src_place) &&  // NOLINT
           platform::is_cpu_place(dst_place)) {
    auto src_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, src_place);
    auto dst_cpu_place = BOOST_GET_CONST(platform::CPUPlace, dst_place);
    memory::Copy(dst_cpu_place, dst_ptr, src_gpu_place, src_ptr, size, nullptr);
  }
  else if (platform::is_cpu_place(src_place) &&  // NOLINT
           platform::is_gpu_place(dst_place)) {
    auto src_cpu_place = BOOST_GET_CONST(platform::CPUPlace, src_place);
    auto dst_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, dst_place);
    memory::Copy(dst_gpu_place, dst_ptr, src_cpu_place, src_ptr, size, nullptr);
  }
  else if (platform::is_gpu_place(src_place) &&  // NOLINT
           platform::is_gpu_place(dst_place)) {
    auto src_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, src_place);
    auto dst_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, dst_place);
    memory::Copy(dst_gpu_place, dst_ptr, src_gpu_place, src_ptr, size, nullptr);
  }
  else if (platform::is_cuda_pinned_place(src_place) &&  // NOLINT
           platform::is_gpu_place(dst_place)) {
    auto src_pinned_place =
        BOOST_GET_CONST(platform::CUDAPinnedPlace, src_place);
    auto dst_gpu_place = BOOST_GET_CONST(platform::CUDAPlace, dst_place);
    memory::Copy(dst_gpu_place, dst_ptr, src_pinned_place, src_ptr, size,
                 nullptr);
  }
  else {  // NOLINT
    PADDLE_THROW(platform::errors::Unimplemented(
        "Copy from %s to %s is not supported.", src_place, dst_place));
  }
#endif
}

template <typename Predicate, typename DevCtx>
struct AnyDTypeVisitor {
  Predicate predicate_;
  const Tensor& tensor_;
  const DevCtx& ctx_;
  Tensor* out_;

  AnyDTypeVisitor(Predicate predicate, const Tensor& tensor, const DevCtx& ctx,
                  Tensor* out)
      : predicate_(predicate), tensor_(tensor), ctx_(ctx), out_(out) {}

  template <typename T>
  void apply() const {
    auto t = EigenVector<T>::Flatten(tensor_);
    auto o = EigenScalar<bool>::From(*out_);
    // return any of predicate_(t) is true.
    o.device(*ctx_.eigen_device()) = predicate_(t).any();
  }
};

template <typename Predicate, typename DevCtx>
inline void AnyImpl(Predicate predicate, const framework::Tensor& tensor,
                    const DevCtx& ctx, framework::Tensor* out) {
  VisitDataType(tensor.type(), AnyDTypeVisitor<Predicate, DevCtx>(
                                   predicate, tensor, ctx, out));
}

template <typename Predicate>
class AnyVisitor : public boost::static_visitor<bool> {
 private:
  const framework::Tensor& tensor_;
  Predicate predicate_;

  bool GetResultHelper(const framework::Tensor& out,
                       const platform::Place& place) const {
    platform::CPUPlace cpu;
    framework::Tensor tmp;
    tmp.Resize({1});
    tmp.mutable_data<bool>(cpu);
    auto ctx = platform::DeviceContextPool::Instance().Get(place);
    ctx->Wait();
    TensorCopy(out, cpu, *ctx, &tmp);
    ctx->Wait();
    return GetResult(tmp, cpu);
  }

 public:
  AnyVisitor(const framework::Tensor& tensor, Predicate predicate)
      : tensor_(tensor), predicate_(std::move(predicate)) {}

  template <typename Place>
  bool operator()(const Place& place) const {
    framework::Tensor out;
    out.Resize({1});
    out.mutable_data<bool>(place);
    auto* ctx = platform::DeviceContextPool::Instance().GetByPlace(place);
    AnyImpl(predicate_, tensor_, *ctx, &out);
    return this->GetResult(out, place);
  }

  bool GetResult(const framework::Tensor& out,
                 const platform::XPUPlace& xpu) const {
    return GetResultHelper(out, xpu);
  }

  bool GetResult(const framework::Tensor& out,
                 const platform::CUDAPlace& gpu) const {
    return GetResultHelper(out, gpu);
  }

  bool GetResult(const framework::Tensor& out,
                 const platform::CPUPlace& cpu) const {
    return *out.data<bool>();
  }

  bool GetResult(const framework::Tensor& out,
                 const platform::CUDAPinnedPlace& cpu) const {
    return *out.data<bool>();
  }
};

template <typename Predicate>
class AnyOutVisitor : public boost::static_visitor<> {
 private:
  const framework::Tensor& tensor_;
  mutable framework::Tensor* out_;
  Predicate predicate_;

 public:
  AnyOutVisitor(const framework::Tensor& tensor, Predicate predicate,
                framework::Tensor* out)
      : tensor_(tensor), out_(out), predicate_(std::move(predicate)) {}

  template <typename Place>
  void operator()(const Place& place) const {
    auto* ctx = platform::DeviceContextPool::Instance().GetByPlace(place);
    out_->Resize({1});
    out_->mutable_data<bool>(place);
    AnyImpl(predicate_, tensor_, *ctx, out_);
  }
};

template <typename Predicate>
inline bool Any(const framework::Tensor& tensor, Predicate predicate) {
  AnyVisitor<Predicate> visitor(tensor, predicate);
  auto place = tensor.place();
  return platform::VisitPlace(place, visitor);
}

template <typename Predicate>
inline void Any(const framework::Tensor& tensor, Predicate predicate,
                framework::Tensor* out) {
  AnyOutVisitor<Predicate> visitor(tensor, predicate, out);
  auto place = tensor.place();
  platform::VisitPlace(place, visitor);
}

template <typename Predicate, typename DevCtx>
struct AllDTypeVisitor {
  Predicate predicate_;
  const Tensor& tensor_;
  const DevCtx& ctx_;
  Tensor* out_;

  AllDTypeVisitor(Predicate predicate, const Tensor& tensor, const DevCtx& ctx,
                  Tensor* out)
      : predicate_(predicate), tensor_(tensor), ctx_(ctx), out_(out) {}

  template <typename T>
  void apply() const {
    auto t = EigenVector<T>::Flatten(tensor_);
    auto o = EigenVector<bool>::Flatten(*out_);
    o.device(*ctx_.eigen_device()) = predicate_(t);
  }
};

template <typename Predicate, typename DevCtx>
inline void AllImpl(Predicate predicate, const framework::Tensor& tensor,
                    const DevCtx& ctx, framework::Tensor* out) {
  VisitDataType(tensor.type(), AllDTypeVisitor<Predicate, DevCtx>(
                                   predicate, tensor, ctx, out));
}

template <typename Predicate>
class AllOutVisitor : public boost::static_visitor<> {
 private:
  const framework::Tensor& tensor_;
  mutable framework::Tensor* out_;
  Predicate predicate_;

 public:
  AllOutVisitor(const framework::Tensor& tensor, Predicate predicate,
                framework::Tensor* out)
      : tensor_(tensor), out_(out), predicate_(predicate) {}

  template <typename Place>
  void operator()(const Place& place) const {
    auto* ctx = platform::DeviceContextPool::Instance().GetByPlace(place);
    out_->Resize(tensor_.dims());
    out_->mutable_data<bool>(place);
    AllImpl(predicate_, tensor_, *ctx, out_);
  }
};

template <typename Predicate>
inline void All(const framework::Tensor& tensor, Predicate predicate,
                framework::Tensor* out) {
  AllOutVisitor<Predicate> visitor(tensor, predicate, out);
  auto place = tensor.place();
  platform::VisitPlace(place, visitor);
}

struct ContainsNANPredicate {
  template <typename T>
  auto operator()(const T& eigen_vec) const
      -> decltype(std::declval<T>().isnan()) {
    // Cast eigen_vector to vector of bool. true if is inf.
    return eigen_vec.isnan();
  }
};

bool TensorContainsNAN(const framework::Tensor& tensor) {
  ContainsNANPredicate predicate;
  return Any(tensor, predicate);
}

void TensorContainsNAN(const framework::Tensor& tensor,
                       framework::Tensor* out) {
  ContainsNANPredicate predicate;
  Any(tensor, predicate, out);
}

void TensorContainsNANV2(const framework::Tensor& tensor,
                         framework::Tensor* out) {
  ContainsNANPredicate predicate;
  All(tensor, predicate, out);
}

struct ContainsInfPredicate {
  template <typename T>
  auto operator()(const T& eigen_vec) const
      -> decltype(std::declval<T>().isinf()) {
    // Cast eigen_vector to vector of bool. true if is inf.
    return eigen_vec.isinf();
  }
};

bool TensorContainsInf(const framework::Tensor& tensor) {
  ContainsInfPredicate predicate;
  return Any(tensor, predicate);
}

void TensorContainsInf(const framework::Tensor& tensor,
                       framework::Tensor* out) {
  ContainsInfPredicate predicate;
  Any(tensor, predicate, out);
}

void TensorContainsInfV2(const framework::Tensor& tensor,
                         framework::Tensor* out) {
  ContainsInfPredicate predicate;
  All(tensor, predicate, out);
}

// NOTE(dzhwinter):
// Isfinite need a AllVisitor to loop through all the elements.
// We choose two cuda call instead of one allvisitor. The AllVisitor
// should be implemented if the performance hurts.
bool TensorIsfinite(const framework::Tensor& tensor) {
  ContainsInfPredicate pred_inf;
  ContainsNANPredicate pred_nan;
  return !Any(tensor, pred_inf) && !Any(tensor, pred_nan);
}

#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
template <typename T>
static inline void __global__ BothFalse(const T* cmp, T* out, int element_num) {
  CUDA_KERNEL_LOOP(i, element_num) { out[i] = (!cmp[i]) && (!out[i]); }
}
#endif

struct BothFalseVisitor : public boost::static_visitor<> {
  const framework::Tensor& in_;
  mutable framework::Tensor* out_;
  BothFalseVisitor(const framework::Tensor& in, framework::Tensor* out)
      : in_(in), out_(out) {}

  template <typename Place>
  void operator()(const Place& place) const {
    VisitorImpl(place);
  }

  void VisitorImpl(const platform::XPUPlace& xpu) const {
    PADDLE_THROW(platform::errors::Unimplemented("XPUPlace is not supported"));
  }

  void VisitorImpl(const platform::CUDAPlace& gpu) const {
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
    auto* ctx = platform::DeviceContextPool::Instance().GetByPlace(gpu);
    constexpr int MAX_BLOCK_DIM = 512;
    const int MAX_GRID_DIM = ctx->GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
    int element_num = in_.numel();
    int block_size = (element_num >= MAX_BLOCK_DIM)
                         ? MAX_BLOCK_DIM
                         : (1 << static_cast<int>(std::log2(element_num)));
    int grid_size = element_num / block_size;
    grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
    BothFalse<bool><<<grid_size, block_size, 0, ctx->stream()>>>(
        in_.data<bool>(), out_->mutable_data<bool>(gpu), element_num);
#endif
  }

  void VisitorImpl(const platform::CPUPlace& cpu) const {
    int num = in_.numel();
    const bool* in_ptr = in_.data<bool>();
    bool* out_ptr = out_->data<bool>();
    for (int i = 0; i < num; ++i) {
      bool lhs = !in_ptr[i];
      bool rhs = !out_ptr[i];
      out_ptr[i] = lhs && rhs;
    }
  }

  void VisitorImpl(
      const platform::CUDAPinnedPlace& cpu /* equals to cpu*/) const {
    int num = in_.numel();
    const bool* in_ptr = in_.data<bool>();
    bool* out_ptr = out_->data<bool>();
    for (int i = 0; i < num; ++i) {
      bool lhs = !in_ptr[i];
      bool rhs = !out_ptr[i];
      out_ptr[i] = lhs && rhs;
    }
  }
};

void TensorIsfinite(const framework::Tensor& tensor, framework::Tensor* out) {
  framework::Tensor tmp;
  TensorContainsInf(tensor, &tmp);
  TensorContainsNAN(tensor, out);
  BothFalseVisitor visitor(tmp, out);
  auto place = tensor.place();
  platform::VisitPlace(place, visitor);
}

void TensorIsfiniteV2(const framework::Tensor& tensor, framework::Tensor* out) {
  framework::Tensor tmp;
  TensorContainsInfV2(tensor, &tmp);
  TensorContainsNANV2(tensor, out);
  BothFalseVisitor visitor(tmp, out);
  auto place = tensor.place();
  platform::VisitPlace(place, visitor);
}

void TensorToStream(std::ostream& os, const Tensor& tensor,
                    const platform::DeviceContext& dev_ctx) {
  {  // the 1st field, uint32_t version
    constexpr uint32_t version = 0;
    os.write(reinterpret_cast<const char*>(&version), sizeof(version));
  }
  {  // the 2nd field, tensor description
     // int32_t  size
     // void*    protobuf message
    proto::VarType::TensorDesc desc;
    desc.set_data_type(tensor.type());
    auto dims = framework::vectorize(tensor.dims());
    auto* pb_dims = desc.mutable_dims();
    pb_dims->Resize(static_cast<int>(dims.size()), 0);
    std::copy(dims.begin(), dims.end(), pb_dims->begin());
    int32_t size = desc.ByteSize();
    os.write(reinterpret_cast<const char*>(&size), sizeof(size));
    auto out = desc.SerializeAsString();
    os.write(out.data(), size);
  }
  {  // the 3rd field, tensor data
    uint64_t size = tensor.numel() * framework::SizeOfType(tensor.type());

    auto* data_ptr = tensor.data<void>();
    PADDLE_ENFORCE_LT(size, (std::numeric_limits<std::streamsize>::max)(),
                      platform::errors::ResourceExhausted(
                          "tensor size %d overflow when writing tensor", size));
    if (platform::is_gpu_place(tensor.place())) {
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
      constexpr size_t kBufSize = 1024 * 1024 * 64;  // 64MB
      std::unique_ptr<char[]> buf(new char[kBufSize]);
      auto& gpu_dev_ctx =
          static_cast<const platform::CUDADeviceContext&>(dev_ctx);
      platform::CPUPlace cpu;
      uintptr_t data = reinterpret_cast<uintptr_t>(data_ptr);
      while (size != 0) {
        size_t size_to_write = std::min(kBufSize, static_cast<size_t>(size));
        memory::Copy(cpu, buf.get(),
                     BOOST_GET_CONST(platform::CUDAPlace, tensor.place()),
                     reinterpret_cast<const void*>(data), size_to_write,
                     gpu_dev_ctx.stream());
        gpu_dev_ctx.Wait();
        os.write(buf.get(), size_to_write);
        data += size_to_write;
        size -= size_to_write;
      }
#else
      PADDLE_THROW(platform::errors::Unimplemented(
          "CUDAPlace is not supported when not compiled with CUDA"));
#endif
    } else if (platform::is_xpu_place(tensor.place())) {
#ifdef PADDLE_WITH_XPU
      constexpr size_t kBufSize = 1024 * 1024 * 64;  // 64MB
      std::unique_ptr<char[]> buf(new char[kBufSize]);
      auto& xpu_dev_ctx =
          static_cast<const platform::XPUDeviceContext&>(dev_ctx);
      platform::CPUPlace cpu;
      uintptr_t data = reinterpret_cast<uintptr_t>(data_ptr);
      while (size != 0) {
        size_t size_to_write = std::min(kBufSize, static_cast<size_t>(size));
        memory::Copy(cpu, buf.get(),
                     BOOST_GET_CONST(platform::XPUPlace, tensor.place()),
                     reinterpret_cast<const void*>(data), size_to_write);
        xpu_dev_ctx.Wait();
        os.write(buf.get(), size_to_write);
        data += size_to_write;
        size -= size_to_write;
      }
#else
      PADDLE_THROW(platform::errors::Unimplemented(
          "XPUPlace is not supported when not compiled with XPU"));
#endif
    } else {
      os.write(static_cast<const char*>(data_ptr),
               static_cast<std::streamsize>(size));
    }
  }
}

struct DeserializedDataFunctor {
  DeserializedDataFunctor(void** buf, Tensor* tensor,
                          const platform::Place& place)
      : buf_(buf), tensor_(tensor), place_(place) {}

  template <typename T>
  void apply() {
    *buf_ = tensor_->mutable_data<T>(place_);
  }

  void** buf_;
  Tensor* tensor_;
  platform::Place place_;
};

void TensorFromStream(std::istream& is, Tensor* tensor,
                      const platform::DeviceContext& dev_ctx,
                      const size_t& seek, const std::vector<int64_t>& shape) {
  uint32_t version;
  is.read(reinterpret_cast<char*>(&version), sizeof(version));

  PADDLE_ENFORCE_EQ(
      version, 0U,
      platform::errors::InvalidArgument(
          "tensor version %u is not supported, Only version 0 is supported",
          version));

  proto::VarType::TensorDesc desc;
  {  // int32_t size
    // proto buffer
    int32_t size;
    is.read(reinterpret_cast<char*>(&size), sizeof(size));
    std::unique_ptr<char[]> buf(new char[size]);
    is.read(reinterpret_cast<char*>(buf.get()), size);
    PADDLE_ENFORCE_EQ(
        desc.ParseFromArray(buf.get(), size), true,
        platform::errors::InvalidArgument("Cannot parse tensor desc"));
  }
  {  // read tensor
    tensor->Resize(framework::make_ddim(shape));
    size_t seekg = seek * framework::SizeOfType(desc.data_type());
    is.seekg(seekg, is.cur);

    void* buf;
    auto ctx = platform::CPUDeviceContext();
    size_t size = tensor->numel() * framework::SizeOfType(desc.data_type());
    if (platform::is_gpu_place(dev_ctx.GetPlace()) ||
        platform::is_xpu_place(dev_ctx.GetPlace())) {
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP) || \
    defined(PADDLE_WITH_XPU)
      Tensor cpu_tensor;
      cpu_tensor.Resize(framework::make_ddim(shape));
      framework::VisitDataType(
          desc.data_type(),
          DeserializedDataFunctor(&buf, &cpu_tensor, ctx.GetPlace()));
      is.read(static_cast<char*>(buf), size);
      auto dst_place = dev_ctx.GetPlace();
      framework::TensorCopy(cpu_tensor, dst_place, dev_ctx, tensor);
#else
      if (platform::is_gpu_place(dev_ctx.GetPlace())) {
        PADDLE_THROW(platform::errors::Unimplemented(
            "CUDAPlace is not supported when not compiled with CUDA"));
      } else {
        PADDLE_THROW(platform::errors::Unimplemented(
            "XPUPlace is not supported when not compiled with XPU"));
      }
#endif
    } else {
      framework::VisitDataType(
          desc.data_type(),
          DeserializedDataFunctor(&buf, tensor, ctx.GetPlace()));
      is.read(static_cast<char*>(buf), size);
    }
  }
}

void TensorFromStream(std::istream& is, Tensor* tensor,
                      const platform::DeviceContext& dev_ctx) {
  uint32_t version;
  is.read(reinterpret_cast<char*>(&version), sizeof(version));
  PADDLE_ENFORCE_EQ(
      version, 0U,
      platform::errors::InvalidArgument(
          "tensor version %u is not supported, Only version 0 is supported",
          version));
  proto::VarType::TensorDesc desc;
  {  // int32_t size
     // proto buffer
    int32_t size;
    is.read(reinterpret_cast<char*>(&size), sizeof(size));
    std::unique_ptr<char[]> buf(new char[size]);
    is.read(reinterpret_cast<char*>(buf.get()), size);
    PADDLE_ENFORCE_EQ(
        desc.ParseFromArray(buf.get(), size), true,
        platform::errors::InvalidArgument("Cannot parse tensor desc"));
  }
  {  // read tensor
    std::vector<int64_t> dims;
    dims.reserve(static_cast<size_t>(desc.dims().size()));
    std::copy(desc.dims().begin(), desc.dims().end(), std::back_inserter(dims));
    tensor->Resize(framework::make_ddim(dims));
    void* buf;
    auto ctx = platform::CPUDeviceContext();
    size_t size = tensor->numel() * framework::SizeOfType(desc.data_type());
    if (platform::is_gpu_place(dev_ctx.GetPlace()) ||
        platform::is_xpu_place(dev_ctx.GetPlace())) {
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP) || \
    defined(PADDLE_WITH_XPU)
      Tensor cpu_tensor;
      cpu_tensor.Resize(framework::make_ddim(dims));
      framework::VisitDataType(
          desc.data_type(),
          DeserializedDataFunctor(&buf, &cpu_tensor, ctx.GetPlace()));
      is.read(static_cast<char*>(buf), size);
      auto dst_place = dev_ctx.GetPlace();
      framework::TensorCopy(cpu_tensor, dst_place, dev_ctx, tensor);
#else
      if (platform::is_gpu_place(dev_ctx.GetPlace())) {
        PADDLE_THROW(platform::errors::Unimplemented(
            "CUDAPlace is not supported when not compiled with CUDA"));
      } else {
        PADDLE_THROW(platform::errors::Unimplemented(
            "XPUPlace is not supported when not compiled with XPU"));
      }
#endif
    } else {
      framework::VisitDataType(
          desc.data_type(),
          DeserializedDataFunctor(&buf, tensor, ctx.GetPlace()));
      is.read(static_cast<char*>(buf), size);
    }
  }
}

// get tensor data point by DLDataType
void* GetDstPtrByDLDataType(DLDataType type, framework::Tensor* dst,
                            const platform::Place& dst_place) {
  // vector types not currently supported
  PADDLE_ENFORCE_LE(type.lanes, 1,
                    platform::errors::Unimplemented(
                        "Vector type is not supported currently."));

  switch (type.bits) {
    case 8:
      if (type.code == kDLInt)
        return static_cast<void*>(dst->mutable_data<int8_t>(dst_place));
      if (type.code == kDLUInt)
        return static_cast<void*>(dst->mutable_data<uint8_t>(dst_place));
      PADDLE_THROW(platform::errors::Unimplemented(
          "DLDataType code <%d> is illegal when DLDataType.bits is <%d>.",
          type.code, type.bits));
    case 16:
      if (type.code == kDLInt)
        return static_cast<void*>(dst->mutable_data<int16_t>(dst_place));
      if (type.code == kDLFloat)
        return static_cast<void*>(
            dst->mutable_data<paddle::platform::float16>(dst_place));
      PADDLE_THROW(platform::errors::Unimplemented(
          "DLDataType code <%d> is illegal when DLDataType.bits is <%d>.",
          type.code, type.bits));
    case 32:
      if (type.code == kDLInt)
        return static_cast<void*>(dst->mutable_data<int32_t>(dst_place));
      if (type.code == kDLFloat)
        return static_cast<void*>(dst->mutable_data<float>(dst_place));
      PADDLE_THROW(platform::errors::Unimplemented(
          "DLDataType code <%d> is illegal when DLDataType.bits is <%d>.",
          type.code, type.bits));
    case 64:
      if (type.code == kDLInt)
        return static_cast<void*>(dst->mutable_data<int64_t>(dst_place));
      if (type.code == kDLFloat)
        return static_cast<void*>(dst->mutable_data<double>(dst_place));
      PADDLE_THROW(platform::errors::Unimplemented(
          "DLDataType code <%d> is illegal when DLDataType.bits is <%d>.",
          type.code, type.bits));
    default:
      PADDLE_THROW(platform::errors::Unimplemented(
          "Unsupported DLDataType.bits %d.", type.bits));
  }
}

void TensorFromDLPack(const ::DLTensor& dl_tensor, framework::Tensor* dst) {
  platform::CPUPlace dst_place = platform::CPUPlace();
  platform::CPUPlace src_place = platform::CPUPlace();

  std::vector<int64_t> vec;
  std::copy(dl_tensor.shape, dl_tensor.shape + dl_tensor.ndim,
            std::back_inserter(vec));

  framework::DDim vddim = framework::make_ddim(vec);

  dst->Resize(vddim);
  ::DLDataType type = dl_tensor.dtype;
  void* dst_ptr = GetDstPtrByDLDataType(type, dst, dst_place);

  auto src_ptr = static_cast<const void*>(dl_tensor.data);
  auto size = paddle::framework::product(vddim) * type.bits / 8;

  if (dl_tensor.ctx.device_type == kDLCPU) {
    memory::Copy(dst_place, dst_ptr, src_place, src_ptr, size);
  }
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
  if (dl_tensor.ctx.device_type == kDLGPU) {
    platform::CUDAPlace dst_place =
        platform::CUDAPlace(dl_tensor.ctx.device_id);
    platform::CUDAPlace src_place =
        platform::CUDAPlace(dl_tensor.ctx.device_id);
    dst_ptr = GetDstPtrByDLDataType(type, dst, dst_place);
    auto* ctx = platform::DeviceContextPool::Instance().GetByPlace(dst_place);
    memory::Copy(
        dst_place, dst_ptr, src_place, src_ptr, size,
        reinterpret_cast<const platform::CUDADeviceContext&>(*ctx).stream());
  }
#endif
#ifdef PADDLE_WITH_XPU
  PADDLE_THROW(platform::errors::Unimplemented("XPUPlace is not supported"));
#endif
}

template <typename T>
std::string format_tensor(const framework::Tensor& tensor) {
  // TODO(zhiqiu): use the print option to format tensor.
  return "NOT IMPLEMENTED";
}

template <typename T>
std::ostream& print_tensor(std::ostream& os, const framework::Tensor& tensor) {
  auto inspect = tensor.data<T>();
  auto element_num = tensor.numel();

  os << "  - data: [";
  // Note: int8_t && uint8_t is typedf of char, ostream unable to print properly
  if (typeid(int8_t) == typeid(T) || typeid(uint8_t) == typeid(T)) {
    if (element_num > 0) {
      os << signed(inspect[0]);
      for (int j = 1; j < element_num; ++j) {
        os << " " << signed(inspect[j]);
      }
    }
  } else {
    if (element_num > 0) {
      os << inspect[0];
      for (int j = 1; j < element_num; ++j) {
        os << " " << inspect[j];
      }
    }
  }
  os << "]";
  return os;
}

template <>
std::ostream& print_tensor<paddle::platform::complex64>(
    std::ostream& os, const framework::Tensor& tensor) {
  auto inspect = tensor.data<paddle::platform::complex64>();
  auto element_num = tensor.numel();

  os << "  - data: [";
  if (element_num > 0) {
    os << signed(inspect[0].real) << "+" << signed(inspect[0].imag) << "j";
    for (int j = 1; j < element_num; ++j) {
      os << " " << signed(inspect[j].real) << "+" << signed(inspect[j].imag)
         << "j";
    }
  }
  os << "]";
  return os;
}

template <>
std::ostream& print_tensor<paddle::platform::complex128>(
    std::ostream& os, const framework::Tensor& tensor) {
  auto inspect = tensor.data<paddle::platform::complex128>();
  auto element_num = tensor.numel();

  os << "  - data: [";
  if (element_num > 0) {
    os << signed(inspect[0].real) << "+" << signed(inspect[0].imag) << "j";
    for (int j = 1; j < element_num; ++j) {
      os << " " << signed(inspect[j].real) << "+" << signed(inspect[j].imag)
         << "j";
    }
  }
  os << "]";
  return os;
}

std::ostream& operator<<(std::ostream& os, const Tensor& t) {
  os << "  - place: " << t.place() << "\n";
  os << "  - shape: [" << t.dims() << "]\n";
  os << "  - layout: " << DataLayoutToString(t.layout()) << "\n";

  Tensor tensor;
  tensor.Resize(t.dims());
  if (platform::is_cpu_place(t.place())) {
    tensor.ShareDataWith(t);
  } else {
    platform::CPUPlace place;
    framework::TensorCopy(t, place, &tensor);
    platform::DeviceContextPool& pool = platform::DeviceContextPool::Instance();
    auto& dev_ctx = *pool.Get(t.place());
    dev_ctx.Wait();
  }

#define PrintTensorCallback(cpp_type, proto_type) \
  do {                                            \
    if (tensor.type() == proto_type) {            \
      os << "  - dtype: " << proto_type << "\n";  \
      print_tensor<cpp_type>(os, tensor);         \
      return os;                                  \
    }                                             \
  } while (0)

  _ForEachDataType_(PrintTensorCallback);
  VLOG(1) << "PrintVar: unrecognized data type:" << t.type();
  return os;
}

}  // namespace framework
}  // namespace paddle
