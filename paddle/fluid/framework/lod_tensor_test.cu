#include "hip/hip_runtime.h"
//   Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "gtest/gtest.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/init.h"
#include "paddle/fluid/platform/place.h"

__global__ void test(size_t* a, int size) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    a[i] *= 2;
  }
}

TEST(LoD, data) {
  paddle::framework::InitDevices(true);

  paddle::framework::LoD lod{{0, 1, 2}};
  lod.push_back({0, 2, 4, 5});
  lod.push_back(std::vector<size_t>({0, 1, 6, 8, 10, 11}));

  auto& v = lod[0];
  paddle::platform::CUDAPlace gpu(0);
  test<<<1, 1>>>(v.CUDAMutableData(gpu), v.size());
  hipDeviceSynchronize();
  for (size_t i = 0; i < v.size(); ++i) {
    EXPECT_EQ(v[i], i * 2);
  }
}

TEST(LoDTensor, LoDInGPU) {
  paddle::framework::InitDevices(true);

  paddle::framework::LoDTensor lod_tensor;
  paddle::platform::CUDAPlace place(0);

  paddle::framework::LoD src_lod;
  src_lod.push_back(std::vector<size_t>{0, 2, 4, 6, 8, 10, 12, 14});

  lod_tensor.Resize({14, 16});
  lod_tensor.mutable_data<float>(place);

  lod_tensor.set_lod(src_lod);
  EXPECT_EQ(lod_tensor.lod_element(0, 2).first, 4UL);
  EXPECT_EQ(lod_tensor.lod_element(0, 4).first, 8UL);

  auto lod = lod_tensor.lod();

  test<<<1, 8>>>(lod[0].CUDAMutableData(place), lod[0].size());
  hipDeviceSynchronize();

  for (size_t i = 0; i < src_lod[0].size(); ++i) {
    EXPECT_EQ(lod[0].data()[i], src_lod[0].data()[i] * 2);
  }
}
