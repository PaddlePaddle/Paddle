#include "hip/hip_runtime.h"
//  Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include <thrust/device_vector.h>
#include <sstream>

#include "gtest/gtest.h"
#include "paddle/fluid/framework/dim.h"

__global__ void test(paddle::framework::Dim<2>* o) {
  o[0] = paddle::framework::make_dim(5, 6);
}

__global__ void dyn_idx_gpu(int64_t* o) {
  auto d = paddle::framework::make_dim(5, 6);
  o[0] = d[1];
}

TEST(Dim, Equality) {
  // construct a Dim on the CPU
  auto a = paddle::framework::make_dim(3, 4);
  EXPECT_EQ(a[0], 3);
  EXPECT_EQ(a[1], 4);

  // construct a Dim on the GPU
  thrust::device_vector<paddle::framework::Dim<2>> t(2);
  test<<<1, 1>>>(thrust::raw_pointer_cast(t.data()));
  a = t[0];
  EXPECT_EQ(a[0], 5);
  EXPECT_EQ(a[1], 6);

  // product
  EXPECT_EQ(paddle::framework::product(a), 30);

  // mutate a Dim
  auto b = paddle::framework::make_dim(7, 8);
  b[1] = 10;
  EXPECT_EQ(b[0], 7);
  EXPECT_EQ(b[1], 10);

  b[0] = 8;
  b[1] = 11;
  EXPECT_EQ(b[0], 8);
  EXPECT_EQ(b[1], 11);

  // dynamic access on GPU
  thrust::device_vector<int64_t> r(1);
  dyn_idx_gpu<<<1, 1>>>(thrust::raw_pointer_cast(r.data()));
  int64_t res = r[0];
  EXPECT_EQ(res, 6);
}

TEST(Dim, Bool) {
  auto a = paddle::framework::make_dim(3, 4);
  auto b = paddle::framework::make_dim(5, 6);
  auto c = paddle::framework::make_dim(3, 4);

  // comparison
  EXPECT_TRUE(a == a);
  EXPECT_FALSE(a == b);
  EXPECT_TRUE(a == c);
}

TEST(Dim, Print) {
  {
    std::stringstream ss;
    auto a = paddle::framework::make_dim(2, 3);
    ss << a;
    EXPECT_EQ(ss.str(), "2, 3");
  }
  {
    std::stringstream ss;
    ss << paddle::framework::make_dim(8);
    EXPECT_EQ(ss.str(), "8");
  }
}
