#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/platform/bfloat16.h"

#define GLOG_NO_ABBREVIATED_SEVERITIES  // msvc conflict logging with windows.h
#include <glog/logging.h>
#include <gtest/gtest.h>
#include <iostream>
#include "paddle/fluid/framework/lod_tensor.h"

#if defined(PADDLE_CUDA_BF16)
namespace paddle {
namespace platform {

TEST(bfloat16, convert_float32_to_bfloat16_on_gpu) {
  // Convert float32 to bfloat16
  EXPECT_EQ((bfloat16(1.0f)).x, 0x3f80);
  EXPECT_EQ((bfloat16(0.5f)).x, 0x3f00);
  EXPECT_EQ((bfloat16(0.33333f)).x, 0x3eab);
  EXPECT_EQ((bfloat16(0.0f)).x, 0x0000);
  EXPECT_EQ((bfloat16(-0.0f)).x, 0x8000);
  EXPECT_EQ((bfloat16(65536.0f)).x, 0x4780);
}

TEST(bfloat16, assignment_operator_on_gpu) {
  // Assignment operator
  bfloat16 v_assign;
  v_assign = hip_bfloat16(bfloat16(1.0f));
  EXPECT_EQ(v_assign.x, 0x3f80);
  v_assign = 0.33333;
  EXPECT_EQ(v_assign.x, 0x3eab);
}

TEST(bfloat16, convert_bfloat16_to_float32_on_gpu) {
  // Conversion operator
  EXPECT_EQ(static_cast<float>(bfloat16(0.5f)), 0.5f);
  EXPECT_NEAR(static_cast<double>(bfloat16(0.33333)), 0.33333, 0.01);
  EXPECT_EQ(static_cast<int>(bfloat16(-1)), -1);
  EXPECT_EQ(static_cast<bool>(bfloat16(true)), true);
}

TEST(bfloat16, lod_tensor_on_gpu) {
  framework::LoDTensor src_tensor;
  framework::LoDTensor gpu_tensor;
  framework::LoDTensor dst_tensor;

  bfloat16 *src_ptr = src_tensor.mutable_data<bfloat16>(
      framework::make_ddim({2, 2}), CPUPlace());

  bfloat16 arr[4] = {bfloat16(1.0f), bfloat16(0.5f), bfloat16(0.33333f),
                     bfloat16(0.0f)};
  memcpy(src_ptr, arr, 4 * sizeof(bfloat16));

  // CPU LoDTensor to GPU LoDTensor
  CUDAPlace gpu_place(0);
  CUDADeviceContext gpu_ctx(gpu_place);
  framework::TensorCopy(src_tensor, gpu_place, gpu_ctx, &gpu_tensor);

  // GPU LoDTensor to CPU LoDTensor
  framework::TensorCopy(gpu_tensor, CPUPlace(), gpu_ctx, &dst_tensor);

  // Sync before comparing LoDTensors
  gpu_ctx.Wait();
  const bfloat16 *dst_ptr = dst_tensor.data<bfloat16>();
  ASSERT_NE(src_ptr, dst_ptr);
  for (size_t i = 0; i < 4; ++i) {
    EXPECT_EQ(src_ptr[i].x, dst_ptr[i].x);
  }
}

TEST(bfloat16, isinf) {
  bfloat16 a;
  a.x = 0x7f80;
  bfloat16 b = bfloat16(INFINITY);
  bfloat16 c = static_cast<bfloat16>(INFINITY);
  EXPECT_EQ(std::isinf(a), true);
  EXPECT_EQ(std::isinf(b), true);
  EXPECT_EQ(std::isinf(c), true);
}

TEST(bfloat16, isnan) {
  bfloat16 a;
  a.x = 0x7fff;
  bfloat16 b = bfloat16(NAN);
  bfloat16 c = static_cast<bfloat16>(NAN);
  EXPECT_EQ(std::isnan(a), true);
  EXPECT_EQ(std::isnan(b), true);
  EXPECT_EQ(std::isnan(c), true);
}

TEST(bfloat16, cast) {
  bfloat16 a;
  a.x = 0x0070;
  auto b = a;
  {
    // change semantic, keep the same value
    bfloat16 c = reinterpret_cast<bfloat16 &>(reinterpret_cast<unsigned &>(b));
    EXPECT_EQ(b, c);
  }

  {
    // use uint32 low 16 bit store float16
    uint32_t c = reinterpret_cast<uint32_t &>(b);
    bfloat16 d;
    d.x = c;
    EXPECT_EQ(b, d);
  }
}

}  // namespace platform
}  // namespace paddle
#endif
