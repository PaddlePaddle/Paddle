#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/platform/bfloat16.h"

#define GLOG_NO_ABBREVIATED_SEVERITIES  // msvc conflict logging with windows.h
#include <glog/logging.h>
#include <gtest/gtest.h>
#include <iostream>

#ifdef PADDLE_CUDA_BF16
using bf16 = __hip_bfloat16;

#define ARITHMETIC_KERNEL(op_type, sign)                                 \
  __global__ void op_type(const bf16 *in1, const bf16 *in2, bf16 *out) { \
    out[0] = in1[0] sign in2[0];                                         \
  }

#define COMPOUND_KERNEL(op_type, sign) \
  __global__ void op_type(bf16 *in1, const bf16 *in2) { in1[0] sign in2[0]; }

#define COMPARISON_KERNEL(op_type, sign)                                 \
  __global__ void op_type(const bf16 *in1, const bf16 *in2, bool *out) { \
    out[0] = in1[0] sign in2[0];                                         \
  }

#define ARITHMETIC_KERNEL_LAUNCH(op_type)                     \
  void Test##op_type(float v_in1, float v_in2, float v_out) { \
    LOG(INFO) << "Test " << #op_type << " on GPU!";           \
    bf16 *in1, *in2, *out;                                    \
    bf16 *d_in1, *d_in2, *d_out;                              \
    int size = sizeof(bf16);                                  \
    hipMalloc(reinterpret_cast<void **>(&d_in1), size);      \
    hipMalloc(reinterpret_cast<void **>(&d_in2), size);      \
    hipMalloc(reinterpret_cast<void **>(&d_out), size);      \
    in1 = reinterpret_cast<bf16 *>(malloc(size));             \
    in2 = reinterpret_cast<bf16 *>(malloc(size));             \
    out = reinterpret_cast<bf16 *>(malloc(size));             \
    in1[0] = bf16(bfloat16(v_in1));                           \
    in2[0] = bf16(bfloat16(v_in2));                           \
    hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);     \
    hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);     \
    op_type<<<1, 1>>>(d_in1, d_in2, d_out);                   \
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);     \
    EXPECT_EQ(static_cast<float>(bfloat16(out[0])), v_out);   \
    free(in1);                                                \
    free(in2);                                                \
    free(out);                                                \
    hipFree(d_in1);                                          \
    hipFree(d_in2);                                          \
    hipFree(d_out);                                          \
  }

#define COMPOUND_KERNEL_LAUNCH(op_type)                       \
  void Test##op_type(float v_in1, float v_in2, float v_out) { \
    LOG(INFO) << "Test " << #op_type << " on GPU!";           \
    bf16 *in1, *in2;                                          \
    bf16 *d_in1, *d_in2;                                      \
    int size = sizeof(bf16);                                  \
    hipMalloc(reinterpret_cast<void **>(&d_in1), size);      \
    hipMalloc(reinterpret_cast<void **>(&d_in2), size);      \
    in1 = reinterpret_cast<bf16 *>(malloc(size));             \
    in2 = reinterpret_cast<bf16 *>(malloc(size));             \
    in1[0] = bf16(bfloat16(v_in1));                           \
    in2[0] = bf16(bfloat16(v_in2));                           \
    hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);     \
    hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);     \
    op_type<<<1, 1>>>(d_in1, d_in2);                          \
    hipMemcpy(in1, d_in1, size, hipMemcpyDeviceToHost);     \
    EXPECT_EQ(static_cast<float>(bfloat16(in1[0])), v_out);   \
    free(in1);                                                \
    free(in2);                                                \
    hipFree(d_in1);                                          \
    hipFree(d_in2);                                          \
  }

#define COMPARISON_KERNEL_LAUNCH(op_type)                    \
  void Test##op_type(float v_in1, float v_in2, bool v_out) { \
    LOG(INFO) << "Test " << #op_type << " on GPU!";          \
    bf16 *in1, *in2;                                         \
    bf16 *d_in1, *d_in2;                                     \
    bool *out, *d_out;                                       \
    int size = sizeof(bf16);                                 \
    hipMalloc(reinterpret_cast<void **>(&d_in1), size);     \
    hipMalloc(reinterpret_cast<void **>(&d_in2), size);     \
    hipMalloc(reinterpret_cast<void **>(&d_out), 1);        \
    in1 = reinterpret_cast<bf16 *>(malloc(size));            \
    in2 = reinterpret_cast<bf16 *>(malloc(size));            \
    out = reinterpret_cast<bool *>(malloc(1));               \
    in1[0] = bf16(bfloat16(v_in1));                          \
    in2[0] = bf16(bfloat16(v_in2));                          \
    hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);    \
    hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);    \
    op_type<<<1, 1>>>(d_in1, d_in2, d_out);                  \
    hipMemcpy(out, d_out, 1, hipMemcpyDeviceToHost);       \
    EXPECT_EQ(out[0], v_out);                                \
    free(in1);                                               \
    free(in2);                                               \
    free(out);                                               \
    hipFree(d_in1);                                         \
    hipFree(d_in2);                                         \
    hipFree(d_out);                                         \
  }

namespace paddle {
namespace platform {

// Arithmetic operations testing
ARITHMETIC_KERNEL(Add, +)
ARITHMETIC_KERNEL(Sub, -)
ARITHMETIC_KERNEL(Mul, *)
ARITHMETIC_KERNEL(Div, /)

ARITHMETIC_KERNEL_LAUNCH(Add)
ARITHMETIC_KERNEL_LAUNCH(Sub)
ARITHMETIC_KERNEL_LAUNCH(Mul)
ARITHMETIC_KERNEL_LAUNCH(Div)

__global__ void Neg(bf16 *in) { in[0] = -in[0]; }

void TestNeg(float v_in, float v_out) {
  LOG(INFO) << "Test Neg on GPU!";
  bf16 *in, *d_in;
  int size = sizeof(bf16);
  hipMalloc(reinterpret_cast<void **>(&d_in), size);
  in = reinterpret_cast<bf16 *>(malloc(size));
  in[0] = bf16(bfloat16(v_in));
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  Neg<<<1, 1>>>(d_in);
  hipMemcpy(in, d_in, size, hipMemcpyDeviceToHost);
  EXPECT_EQ(static_cast<float>(bfloat16(in[0])), v_out);
  free(in);
  hipFree(d_in);
}

TEST(bfloat16, arithmetic_on_gpu) {
  TestAdd(1, 2, 3);
  TestSub(2, 1, 1);
  TestMul(2, 3, 6);
  TestDiv(6, 2, 3);
  TestNeg(1, -1);
}

// Compound operations testing
COMPOUND_KERNEL(AddAssign, +=)
COMPOUND_KERNEL(SubAssign, -=)
COMPOUND_KERNEL(MulAssign, *=)
COMPOUND_KERNEL(DivAssign, /=)

COMPOUND_KERNEL_LAUNCH(AddAssign)
COMPOUND_KERNEL_LAUNCH(SubAssign)
COMPOUND_KERNEL_LAUNCH(MulAssign)
COMPOUND_KERNEL_LAUNCH(DivAssign)

TEST(float16, compound_on_gpu) {
  TestAddAssign(1, 2, 3);
  TestSubAssign(2, 1, 1);
  TestMulAssign(2, 3, 6);
  TestDivAssign(6, 2, 3);
}

// Comparison operations testing
COMPARISON_KERNEL(Equal, ==)
COMPARISON_KERNEL(NotEqual, !=)
COMPARISON_KERNEL(Less, <)
COMPARISON_KERNEL(LessEqual, <=)
COMPARISON_KERNEL(Greater, >)
COMPARISON_KERNEL(GreaterEqual, >=)

COMPARISON_KERNEL_LAUNCH(Equal)
COMPARISON_KERNEL_LAUNCH(NotEqual)
COMPARISON_KERNEL_LAUNCH(Less)
COMPARISON_KERNEL_LAUNCH(LessEqual)
COMPARISON_KERNEL_LAUNCH(Greater)
COMPARISON_KERNEL_LAUNCH(GreaterEqual)

TEST(float16, comparision_on_gpu) {
  TestEqual(1, 1, true);
  TestEqual(1, 2, false);
  TestNotEqual(2, 3, true);
  TestNotEqual(2, 2, false);
  TestLess(3, 4, true);
  TestLess(3, 3, false);
  TestLessEqual(3, 3, true);
  TestLessEqual(3, 2, false);
  TestGreater(4, 3, true);
  TestGreater(4, 4, false);
  TestGreaterEqual(4, 4, true);
  TestGreaterEqual(4, 5, false);
}

}  // namespace platform
}  // namespace paddle
#endif  // PADDLE_CUDA_BF16
