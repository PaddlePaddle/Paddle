/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <vector>

#include "glog/logging.h"
#include "gtest/gtest.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/memory/allocation/allocator_facade.h"
#include "paddle/phi/core/platform/device_context.h"

TEST(Device, Init) {
  using phi::DeviceContext;
  using phi::GPUContext;
  using phi::GPUPlace;

  int count = paddle::platform::GetGPUDeviceCount();
  for (int i = 0; i < count; i++) {
    phi::GPUContext* device_context = new phi::GPUContext(GPUPlace(i));
    device_context->SetAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(GPUPlace(i), device_context->stream())
            .get());
    device_context->SetHostAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(phi::CPUPlace())
            .get());
    device_context->SetZeroAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetZeroAllocator(GPUPlace(i))
            .get());
    device_context->SetHostZeroAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetZeroAllocator(phi::CPUPlace())
            .get());
    device_context->SetPinnedAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(phi::GPUPinnedPlace())
            .get());
    device_context->PartialInitWithAllocator();

    Eigen::GpuDevice* gpu_device = device_context->eigen_device();
    ASSERT_NE(nullptr, gpu_device);
    delete device_context;
  }
}

TEST(Device, GPUContext) {
  using phi::GPUContext;
  using phi::GPUPlace;

  int count = paddle::platform::GetGPUDeviceCount();
  for (int i = 0; i < count; i++) {
    phi::GPUContext* device_context = new phi::GPUContext(GPUPlace(i));
    device_context->SetAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(GPUPlace(i), device_context->stream())
            .get());
    device_context->SetHostAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(phi::CPUPlace())
            .get());
    device_context->SetZeroAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetZeroAllocator(GPUPlace(i))
            .get());
    device_context->SetHostZeroAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetZeroAllocator(phi::CPUPlace())
            .get());
    device_context->SetPinnedAllocator(
        paddle::memory::allocation::AllocatorFacade::Instance()
            .GetAllocator(phi::GPUPinnedPlace())
            .get());
    device_context->PartialInitWithAllocator();
    Eigen::GpuDevice* gpu_device = device_context->eigen_device();
    ASSERT_NE(nullptr, gpu_device);
#ifdef PADDLE_WITH_HIP
    miopenHandle_t cudnn_handle = device_context->cudnn_handle();
#else
    hipdnnHandle_t cudnn_handle = device_context->cudnn_handle();
#endif
    ASSERT_NE(nullptr, cudnn_handle);
#ifdef PADDLE_WITH_HIP
    rocblas_handle cublas_handle = device_context->cublas_handle();
#else
    hipblasHandle_t cublas_handle = device_context->cublas_handle();
#endif
    ASSERT_NE(nullptr, cublas_handle);
    delete device_context;
  }
}

TEST(Device, HostZeroAllocator) {
  using phi::GPUPlace;

  auto device_context = std::make_unique<phi::GPUContext>(GPUPlace(0));
  device_context->SetAllocator(
      paddle::memory::allocation::AllocatorFacade::Instance()
          .GetAllocator(GPUPlace(0), device_context->stream())
          .get());
  device_context->SetHostAllocator(
      paddle::memory::allocation::AllocatorFacade::Instance()
          .GetAllocator(phi::CPUPlace())
          .get());
  device_context->SetZeroAllocator(
      paddle::memory::allocation::AllocatorFacade::Instance()
          .GetZeroAllocator(GPUPlace(0))
          .get());
  device_context->SetHostZeroAllocator(
      paddle::memory::allocation::AllocatorFacade::Instance()
          .GetZeroAllocator(phi::CPUPlace())
          .get());
  device_context->SetPinnedAllocator(
      paddle::memory::allocation::AllocatorFacade::Instance()
          .GetAllocator(phi::GPUPinnedPlace())
          .get());
  device_context->PartialInitWithAllocator();

  phi::DenseTensor tensor;
  tensor.Resize({0});
  device_context->HostAlloc<float>(&tensor);
  ASSERT_EQ(tensor.place().GetType(), phi::AllocationType::CPU);
  ASSERT_EQ(tensor.numel(), 0);
  ASSERT_EQ(tensor.dtype(), phi::DataType::FLOAT32);

  phi::GPUContext gpu_context(GPUPlace(0));
  gpu_context.SetHostZeroAllocator(&device_context->GetHostZeroAllocator());
  gpu_context.HostAlloc<float>(&tensor);
  ASSERT_EQ(tensor.place().GetType(), phi::AllocationType::CPU);
}

TEST(Device, DeviceContextPool) {
  using phi::CPUPlace;
  using phi::DeviceContextPool;
  using phi::GPUContext;
  using phi::GPUPlace;
  using phi::Place;

  DeviceContextPool& pool = DeviceContextPool::Instance();
  auto cpu_dev_ctx1 = pool.Get(CPUPlace());
  auto cpu_dev_ctx2 = pool.Get(CPUPlace());
  ASSERT_EQ(cpu_dev_ctx2, cpu_dev_ctx1);

  std::vector<Place> gpu_places;
  int count = paddle::platform::GetGPUDeviceCount();
  for (int i = 0; i < count; ++i) {
    auto dev_ctx = pool.Get(GPUPlace(i));
    ASSERT_NE(dev_ctx, nullptr);
  }
}
