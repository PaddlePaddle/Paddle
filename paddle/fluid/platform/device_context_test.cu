/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/fluid/platform/device_context.h"

#include <iostream>
#include <vector>

#include "glog/logging.h"
#include "gtest/gtest.h"

TEST(Device, Init) {
  using paddle::platform::DeviceContext;
  using paddle::platform::CUDADeviceContext;
  using paddle::platform::CUDAPlace;

  VLOG(3) << "before Init";
  int count = paddle::platform::GetCUDADeviceCount();
  for (int i = 0; i < count; i++) {
    CUDADeviceContext* device_context = new CUDADeviceContext(CUDAPlace(i));
    Eigen::GpuDevice* gpu_device = device_context->eigen_device();
    ASSERT_NE(nullptr, gpu_device);
    delete device_context;
  }
  VLOG(3) << "eigen pass";
}

TEST(Device, CUDADeviceContext) {
  using paddle::platform::CUDADeviceContext;
  using paddle::platform::CUDAPlace;

  VLOG(3) << "cudnn start";
  int count = paddle::platform::GetCUDADeviceCount();
  for (int i = 0; i < count; i++) {
    CUDADeviceContext* device_context = new CUDADeviceContext(CUDAPlace(i));
    VLOG(3) << "device context start";
    Eigen::GpuDevice* gpu_device = device_context->eigen_device();
    ASSERT_NE(nullptr, gpu_device);
    hipdnnHandle_t cudnn_handle = device_context->cudnn_handle();
    VLOG(3) << "cudnn pass";
    ASSERT_NE(nullptr, cudnn_handle);
    hipblasHandle_t cublas_handle = device_context->cublas_handle();
    VLOG(3) << "cublas pass";
    ASSERT_NE(nullptr, cublas_handle);
    ASSERT_NE(nullptr, device_context->stream());
    delete device_context;
  }
}

TEST(Device, DeviceContextPool) {
  using paddle::platform::DeviceContextPool;
  using paddle::platform::CUDADeviceContext;
  using paddle::platform::Place;
  using paddle::platform::CPUPlace;
  using paddle::platform::CUDAPlace;

  VLOG(3) << "before instance";
  DeviceContextPool& pool = DeviceContextPool::Instance();
  VLOG(3) << "after instance";
  auto cpu_dev_ctx1 = pool.Get(CPUPlace());
  auto cpu_dev_ctx2 = pool.Get(CPUPlace());
  ASSERT_EQ(cpu_dev_ctx2, cpu_dev_ctx1);

  std::vector<Place> gpu_places;
  int count = paddle::platform::GetCUDADeviceCount();
  for (int i = 0; i < count; ++i) {
    auto dev_ctx = pool.Get(CUDAPlace(i));
    ASSERT_NE(dev_ctx, nullptr);
  }
}
