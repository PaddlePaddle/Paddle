/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <gtest/gtest.h>
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/memory/memory.h"
#include "paddle/fluid/platform/hostdevice.h"
#include "paddle/fluid/platform/transform.h"

template <typename T>
class Scale {
 public:
  explicit Scale(const T& scale) : scale_(scale) {}
  HOSTDEVICE T operator()(const T& a) const { return a * scale_; }

 private:
  T scale_;
};

template <typename T>
class Multiply {
 public:
  HOSTDEVICE T operator()(const T& a, const T& b) const { return a * b; }
};

using paddle::memory::Alloc;
using paddle::memory::Copy;

using paddle::platform::CPUPlace;
using paddle::platform::CUDAPlace;
using paddle::platform::CPUDeviceContext;
using paddle::platform::CUDADeviceContext;

using paddle::platform::Transform;

TEST(Transform, CPUUnary) {
  CPUDeviceContext ctx;
  float buf[4] = {0.1, 0.2, 0.3, 0.4};
  Transform<CPUDeviceContext> trans;
  trans(ctx, buf, buf + 4, buf, Scale<float>(10));
  for (int i = 0; i < 4; ++i) {
    ASSERT_NEAR(buf[i], static_cast<float>(i + 1), 1e-5);
  }
}

TEST(Transform, GPUUnary) {
  CUDAPlace gpu0(0);
  CUDADeviceContext ctx(gpu0);
  float cpu_buf[4] = {0.1, 0.2, 0.3, 0.4};
  auto gpu_allocation = Alloc(gpu0, sizeof(float) * 4);
  float* gpu_buf = static_cast<float*>(gpu_allocation->ptr());
  Copy(gpu0, gpu_buf, CPUPlace(), cpu_buf, sizeof(cpu_buf), ctx.stream());
  Transform<CUDADeviceContext> trans;
  trans(ctx, gpu_buf, gpu_buf + 4, gpu_buf, Scale<float>(10));
  ctx.Wait();
  Copy(CPUPlace(), cpu_buf, gpu0, gpu_buf, sizeof(cpu_buf), ctx.stream());
  for (int i = 0; i < 4; ++i) {
    ASSERT_NEAR(cpu_buf[i], static_cast<float>(i + 1), 1e-5);
  }
}

TEST(Transform, CPUBinary) {
  int buf[4] = {1, 2, 3, 4};
  Transform<CPUDeviceContext> trans;
  CPUDeviceContext ctx;
  trans(ctx, buf, buf + 4, buf, buf, Multiply<int>());
  for (int i = 0; i < 4; ++i) {
    ASSERT_EQ((i + 1) * (i + 1), buf[i]);
  }
}

TEST(Transform, GPUBinary) {
  int buf[4] = {1, 2, 3, 4};
  CUDAPlace gpu0(0);
  CUDADeviceContext ctx(gpu0);
  auto gpu_allocation = Alloc(gpu0, sizeof(buf));
  int* gpu_buf = static_cast<int*>(gpu_allocation->ptr());
  Copy(gpu0, gpu_buf, CPUPlace(), buf, sizeof(buf), ctx.stream());
  Transform<CUDADeviceContext> trans;
  trans(ctx, gpu_buf, gpu_buf + 4, gpu_buf, gpu_buf, Multiply<int>());
  ctx.Wait();
  Copy(CPUPlace(), buf, gpu0, gpu_buf, sizeof(buf), ctx.stream());
  for (int i = 0; i < 4; ++i) {
    ASSERT_EQ((i + 1) * (i + 1), buf[i]);
  }
}
