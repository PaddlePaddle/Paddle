#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/platform/profiler.h"

#include <hip/hip_runtime.h>

namespace paddle {
namespace platform {

__global__ void DummyKernel(int *a) { a[0] = 0; }

static void ForEachDevice(std::function<void(int)> func) {
  auto original_device = GetCurrentDeviceId();
  int count = GetCUDADeviceCount();
  for (int i = 0; i < count; i++) {
    SetDeviceId(i);
    func(i);
  }
  SetDeviceId(original_device);
}

void DummyKernelAndEvent() {
  for (int i = 0; i < 5; i++) {
    ForEachDevice([](int d) {
      CUDADeviceContext *dev_ctx = new CUDADeviceContext(CUDAPlace(d));
      Mark("_cuda_startup_");
      int *ptr;
      PADDLE_ENFORCE(hipMalloc(&ptr, sizeof(int)));
      DummyKernel<<<1, 1, 0, dev_ctx->stream()>>>(ptr);
      dev_ctx->Wait();
      PADDLE_ENFORCE(hipFree(ptr));
      delete dev_ctx;
    });
  }
}

}  // namespace platform
}  // namespace paddle
