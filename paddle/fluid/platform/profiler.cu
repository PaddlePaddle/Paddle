#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <hip/hip_runtime.h>
#include "paddle/fluid/platform/profiler.h"

namespace paddle {
namespace platform {

__global__ void DummyKernel(int *a) { a[0] = 0; }

static void ForEachDevice(std::function<void(int)> func) {
  auto original_device = platform::GetCurrentDeviceId();
  int count = platform::GetCUDADeviceCount();
  for (int i = 0; i < count; i++) {
    platform::SetDeviceId(i);
    func(i);
  }
  platform::SetDeviceId(original_device);
}

void DummyKernelAndEvent() {
  for (int i = 0; i < 5; i++) {
    ForEachDevice([](int d) {
      platform::SetDeviceId(d);
      hipStream_t stream;
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamCreate(&stream));
      Mark("_cuda_startup_");
      int *ptr;
      PADDLE_ENFORCE_CUDA_SUCCESS(hipMalloc(&ptr, sizeof(int)));
      DummyKernel<<<1, 1, 0, stream>>>(ptr);
      PADDLE_ENFORCE_CUDA_SUCCESS(hipStreamSynchronize(stream));
      PADDLE_ENFORCE_CUDA_SUCCESS(hipFree(ptr));
    });
  }
}

}  // namespace platform
}  // namespace paddle
