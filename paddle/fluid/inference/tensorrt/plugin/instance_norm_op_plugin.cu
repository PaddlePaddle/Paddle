// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/instance_norm_op_plugin.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

hipdnnStatus_t convert_trt2cudnn_dtype(nvinfer1::DataType trt_dtype,
                                      hipdnnDataType_t *cudnn_dtype) {
  switch (trt_dtype) {
    case nvinfer1::DataType::kFLOAT:
      *cudnn_dtype = HIPDNN_DATA_FLOAT;
      break;
    case nvinfer1::DataType::kHALF:
      *cudnn_dtype = HIPDNN_DATA_HALF;
      break;
    default:
      return HIPDNN_STATUS_BAD_PARAM;
  }
  return HIPDNN_STATUS_SUCCESS;
}

int InstanceNormPlugin::initialize() TRT_NOEXCEPT { return 0; }

nvinfer1::Dims InstanceNormPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *inputDims, int nbInputs) TRT_NOEXCEPT {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

bool InstanceNormPlugin::supportsFormat(
    nvinfer1::DataType type, nvinfer1::PluginFormat format) const TRT_NOEXCEPT {
  return ((type == nvinfer1::DataType::kFLOAT ||
           type == nvinfer1::DataType::kHALF) &&
          (format == nvinfer1::PluginFormat::kLINEAR));
}

int InstanceNormPlugin::enqueue(int batch_size, const void *const *inputs,
#if IS_TRT_VERSION_LT(8000)
                                void **outputs, void *workspace,
#else
                                void *const *outputs, void *workspace,
#endif
                                hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = this->getInputDims(0);
  int n = batch_size;
  int c = input_dims.d[0];
  int h = input_dims.d[1];
  int w = input_dims.d[2];

  scale_t.Resize(phi::make_ddim({batch_size, c}));
  bias_t.Resize(phi::make_ddim({batch_size, c}));
  int device_id;
  hipGetDevice(&device_id);
  float *scale_d = scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));

  for (int i = 0; i < batch_size; i++) {
    hipMemcpyAsync(scale_d + i * c, scale_.data(), sizeof(float) * c,
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(bias_d + i * c, bias_.data(), sizeof(float) * c,
                    hipMemcpyHostToDevice, stream);
  }
  platform::dynload::hipdnnSetTensor4dDescriptor(
      b_desc_, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, n * c, 1, 1);

  hipdnnDataType_t cudnn_dtype;
  nvinfer1::DataType data_type = getDataType();
  convert_trt2cudnn_dtype(data_type, &cudnn_dtype);
  platform::dynload::hipdnnSetTensor4dDescriptor(x_desc_, HIPDNN_TENSOR_NCHW,
                                                cudnn_dtype, 1, n * c, h, w);
  platform::dynload::hipdnnSetTensor4dDescriptor(y_desc_, HIPDNN_TENSOR_NCHW,
                                                cudnn_dtype, 1, n * c, h, w);
  float alpha = 1;
  float beta = 0;
  platform::dynload::hipdnnSetStream(handle_, stream);

  void const *x_ptr = inputs[0];
  void *y_ptr = outputs[0];
  platform::dynload::hipdnnBatchNormalizationForwardTraining(
      handle_, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT, &alpha, &beta, x_desc_,
      x_ptr, y_desc_, y_ptr, b_desc_, scale_d, bias_d, 1., nullptr, nullptr,
      eps_, nullptr, nullptr);
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
