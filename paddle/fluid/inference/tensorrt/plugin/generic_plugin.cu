// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/generic_plugin.h"
#include "paddle/fluid/framework/framework.pb.h"
#include "paddle/fluid/framework/op_kernel_type.h"
#include "paddle/fluid/framework/phi_utils.h"
#include "paddle/fluid/inference/tensorrt/dynamic_shape_infermeta_registry.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/core/compat/op_utils.h"
#include "paddle/phi/core/kernel_context.h"
#include "paddle/phi/core/kernel_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

void BuildPhiKernelContextAttr(const framework::OpDesc& op_desc,
                               phi::KernelContext* kernel_context,
                               const phi::KernelSignature& signature,
                               const phi::Kernel& phi_kernel) {
  const phi::KernelArgsDef& args_def = phi_kernel.args_def();
  const auto& attr_names = signature.attr_names;
  const auto& attr_defs = args_def.attribute_defs();

  PADDLE_ENFORCE_EQ(
      attr_names.size(),
      attr_defs.size(),
      platform::errors::InvalidArgument(
          "The attr_names.size() should be equal to attr_defs.size()."));

  framework::AttrReader attr_reader(op_desc.GetAttrMap());

  for (size_t k = 0; k < attr_names.size(); ++k) {
    auto attr_name = attr_names[k];
    auto* attr_ptr = attr_reader.GetAttr(attr_name);
    switch (attr_defs[k].type_index) {
      case phi::AttributeType::SCALAR:
        if (attr_ptr) {
          auto& attr = *attr_ptr;
          switch (AttrTypeID(attr)) {
            case framework::proto::AttrType::FLOAT:
              return kernel_context->EmplaceBackAttr(
                  phi::Scalar(PADDLE_GET_CONST(float, attr)));
              break;
            case framework::proto::AttrType::INT:
              return kernel_context->EmplaceBackAttr(
                  phi::Scalar(PADDLE_GET_CONST(int, attr)));
              break;
            case framework::proto::AttrType::STRING:
              return kernel_context->EmplaceBackAttr(
                  phi::Scalar(PADDLE_GET_CONST(std::string, attr)));
              break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` to Scalar when "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        }
        break;

      case phi::AttributeType::INT_ARRAY:
        if (attr_ptr) {
          auto& attr = *attr_ptr;
          switch (AttrTypeID(attr)) {
            case framework::proto::AttrType::INTS:
              kernel_context->EmplaceBackAttr(std::move(
                  phi::IntArray(PADDLE_GET_CONST(std::vector<int32_t>, attr))));
              break;
            case framework::proto::AttrType::LONGS:
              kernel_context->EmplaceBackAttr(std::move(
                  phi::IntArray(PADDLE_GET_CONST(std::vector<int64_t>, attr))));
              break;
            case framework::proto::AttrType::INT:
              kernel_context->EmplaceBackAttr(
                  phi::IntArray({PADDLE_GET_CONST(int, attr)}));
              break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` to IntArray when "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        }
        break;

      case phi::AttributeType::SCALARS:
        if (attr_ptr) {
          auto& attr = *attr_ptr;
          switch (AttrTypeID(attr)) {
            case framework::proto::AttrType::INTS: {
              const auto& vec = PADDLE_GET_CONST(std::vector<int32_t>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            case framework::proto::AttrType::LONGS: {
              const auto& vec = PADDLE_GET_CONST(std::vector<int64_t>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            case framework::proto::AttrType::FLOATS: {
              const auto& vec = PADDLE_GET_CONST(std::vector<float>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            case framework::proto::AttrType::FLOAT64S: {
              const auto& vec = PADDLE_GET_CONST(std::vector<double>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` to vector<Scalar> when "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        }
        break;

      default:
        if (attr_ptr) {
          auto& attr = *attr_ptr;
          switch (attr_defs[k].type_index) {
            case phi::AttributeType::FLOAT32:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(float, attr));
              break;
            case phi::AttributeType::INT32:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(int, attr));
              break;
            case phi::AttributeType::BOOL:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(bool, attr));
              break;
            case phi::AttributeType::INT64:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(int64_t, attr));
              break;
            case phi::AttributeType::INT32S:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<int>, attr));
              break;
            case phi::AttributeType::DATA_TYPE: {
              auto data_type = paddle::framework::TransToPhiDataType(
                  static_cast<framework::proto::VarType::Type>(
                      PADDLE_GET_CONST(int, attr)));
              kernel_context->EmplaceBackAttr(data_type);
            } break;
            case phi::AttributeType::STRING:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::string, attr));
              break;
            case phi::AttributeType::INT64S:
              switch (AttrTypeID(attr)) {
                case framework::proto::AttrType::LONGS:
                  kernel_context->EmplaceBackAttr(
                      PADDLE_GET_CONST(std::vector<int64_t>, attr));
                  break;
                case framework::proto::AttrType::INTS: {
                  const auto& vector_int_attr =
                      PADDLE_GET_CONST(std::vector<int>, attr);
                  const std::vector<int64_t> vector_int64_attr(
                      vector_int_attr.begin(), vector_int_attr.end());
                  kernel_context->EmplaceBackAttr(vector_int64_attr);
                } break;
                default:
                  PADDLE_THROW(platform::errors::Unimplemented(
                      "Unsupported cast op attribute `%s` to vector<int64_t> "
                      "when ProtoAttr2PhiAttr.",
                      attr_name));
              }
              break;
            case phi::AttributeType::FLOAT32S:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<float>, attr));
              break;
            case phi::AttributeType::STRINGS:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<std::string>, attr));
              break;
            case phi::AttributeType::BOOLS:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<bool>, attr));
              break;
            case phi::AttributeType::FLOAT64S:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<double>, attr));
              break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` when construct "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        }
    }
  }
}

template <typename T>
inline std::string vectorToStr(const std::vector<T>& dims) {
  std::stringstream ss;
  for (size_t i = 0; i < dims.size(); i++) {
    ss << dims[i] << " ";
  }
  return ss.str();
}

GenericPlugin::GenericPlugin(
    const paddle::framework::proto::OpDesc& proto_op_desc,
    const InputOutPutVarInfo& in_out_info) {
  // proto_op_desc_
  proto_op_desc_ = proto_op_desc;
  // op_desc_
  op_desc_ = std::move(framework::OpDesc(proto_op_desc_, nullptr));
  // op_meta_data_
  proto_op_desc_.SerializeToString(&op_meta_data_);
  // inputs_data_type_ and outputs_data_type_
  inputs_data_type_ = in_out_info.inputs_data_type;
  outputs_data_type_ = in_out_info.outputs_data_type;
}

GenericPlugin::GenericPlugin(
    const paddle::framework::proto::OpDesc& proto_op_desc,
    const std::vector<int>& inputs_data_type,
    const std::vector<int>& outputs_data_type) {
  // proto_op_desc_
  proto_op_desc_ = proto_op_desc;
  // op_desc_
  op_desc_ = std::move(framework::OpDesc(proto_op_desc_, nullptr));
  // op_meta_data_
  proto_op_desc_.SerializeToString(&op_meta_data_);
  // inputs_data_type_
  inputs_data_type_ = inputs_data_type;
  // outputs_data_type_
  outputs_data_type_ = outputs_data_type;
}

GenericPlugin::GenericPlugin(void const* serialData, size_t serialLength) {
  // inputs_data_type_
  DeserializeValue(&serialData, &serialLength, &inputs_data_type_);
  // outputs_data_type_
  DeserializeValue(&serialData, &serialLength, &outputs_data_type_);
  // op_meta_data
  std::string op_meta_data((char*)(serialData), serialLength);  // NOLINT
  op_meta_data_ = std::move(op_meta_data);
  // proto_op_desc_
  proto_op_desc_.ParseFromString(op_meta_data_);
  // op_desc_
  op_desc_ = std::move(framework::OpDesc(proto_op_desc_, nullptr));
}

int GenericPlugin::getNbOutputs() const TRT_NOEXCEPT {
  int res = 0;
  for (auto& i : op_desc_.Outputs()) {
    if (!i.second.empty()) res += i.second.size();
  }
  return res;
}

int GenericPlugin::getNbInputs() const TRT_NOEXCEPT {
  int res = 0;
  for (auto& i : op_desc_.Inputs()) {
    if (!i.second.empty()) res += i.second.size();
  }
  return res;
}

nvinfer1::IPluginV2DynamicExt* GenericPlugin::clone() const TRT_NOEXCEPT {
  nvinfer1::IPluginV2DynamicExt* plugin =
      new GenericPlugin(proto_op_desc_, inputs_data_type_, outputs_data_type_);
  plugin->initialize();
  return plugin;
}

void GenericPlugin::serialize(void* buffer) const TRT_NOEXCEPT {
  // inputs_data_type_
  SerializeValue(&buffer, inputs_data_type_);
  // outputs_data_type_
  SerializeValue(&buffer, outputs_data_type_);
  // serialize op_meta_data_
  std::memcpy(buffer, op_meta_data_.c_str(), op_meta_data_.size());
  reinterpret_cast<char*&>(buffer) += op_meta_data_.size();
}

bool GenericPlugin::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  return true;
}

nvinfer1::DataType GenericPlugin::getOutputDataType(
    int index,
    const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

int GenericPlugin::initialize() TRT_NOEXCEPT {
  std::string op_type = op_desc_.Type();

  phi::KernelSignature phi_kernel_signature;
  if (phi::OpUtilsMap::Instance().HasArgumentMappingFn(op_type)) {
    const phi::ArgumentMappingFn* argument_mapping_func =
        phi::OpUtilsMap::Instance().GetArgumentMappingFn(op_type);
    PluginArgumentMappingContext argument_mapping_context(&op_desc_);
    phi_kernel_signature = (*argument_mapping_func)(argument_mapping_context);
  } else {
    phi_kernel_signature =
        phi::DefaultKernelSignatureMap::Instance().Get(op_type);
  }

  phi::KernelKey phi_kernel_key(
      phi::Backend::GPU, phi::DataLayout::ANY, phi::DataType::FLOAT32);

  PADDLE_ENFORCE_EQ(
      phi::KernelFactory::Instance().HasCompatiblePhiKernel(op_type),
      true,
      platform::errors::Fatal("%s has no compatible phi kernel!",
                              op_type.c_str()));

  const phi::Kernel& phi_kernel = phi::KernelFactory::Instance().SelectKernel(
      phi_kernel_signature.name, phi_kernel_key);
  phi_kernel_ = &phi_kernel;

  PADDLE_ENFORCE_EQ(phi_kernel_->IsValid(),
                    true,
                    platform::errors::Fatal("%s phi kernel is invalid!.",
                                            phi_kernel_signature.name));

  paddle::platform::DeviceContextPool& pool =
      paddle::platform::DeviceContextPool::Instance();
  platform::CUDAPlace place(platform::GetCurrentDeviceId());
  auto* dev_ctx = static_cast<const phi::GPUContext*>(pool.Get(place));

  phi_kernel_context_ = new phi::KernelContext(dev_ctx);

  BuildPhiKernelContextAttr(
      op_desc_, phi_kernel_context_, phi_kernel_signature, phi_kernel);
  return 0;
}

nvinfer1::DimsExprs GenericPlugin::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs* inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  CHECK(output_index < getNbOutputs());
  auto& dynamic_infermeta_factory = tensorrt::DynamicMetaFnFactory::Instance();
  PADDLE_ENFORCE_EQ(dynamic_infermeta_factory.Contains(op_desc_.Type()),
                    true,
                    platform::errors::InvalidArgument(
                        "The %s op has no dynamic plugin infershape function!",
                        op_desc_.Type().c_str()));

  auto* infershape_func = dynamic_infermeta_factory.Get(op_desc_.Type());
  return infershape_func(
      output_index, inputs, nb_inputs, expr_builder, op_desc_);
}

void GenericPlugin::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* in,
    int nb_inputs,
    const nvinfer1::DynamicPluginTensorDesc* out,
    int nb_outputs) TRT_NOEXCEPT {
  CHECK(phi_kernel_context_);
  CHECK(phi_kernel_);
  CHECK(nb_inputs == getNbInputs());
  CHECK(nb_outputs == getNbOutputs());
}

int GenericPlugin::enqueue(const nvinfer1::PluginTensorDesc* input_desc,
                           const nvinfer1::PluginTensorDesc* output_desc,
                           const void* const* inputs,
                           void* const* outputs,
                           void* workspace,
                           hipStream_t stream) TRT_NOEXCEPT {
  platform::CUDAPlace place(platform::GetCurrentDeviceId());
  std::vector<phi::DenseTensor>* dense_tensor_inputs =
      new std::vector<phi::DenseTensor>(getNbInputs());
  std::vector<phi::DenseTensor>* dense_tensor_outputs =
      new std::vector<phi::DenseTensor>(getNbOutputs());

  // [TODO]now generic plugin do not support FP16 and INT8 precision
  auto protoType2PhiType = [](int proto_type) -> phi::DataType {
    if (proto_type ==
        static_cast<int>(framework::proto::VarType_Type::VarType_Type_FP32))
      return phi::DataType::FLOAT32;
    else if (proto_type ==
             static_cast<int>(
                 framework::proto::VarType_Type::VarType_Type_INT64))
      return phi::DataType::INT32;
    else if (proto_type ==
             static_cast<int>(
                 framework::proto::VarType_Type::VarType_Type_BOOL))
      return phi::DataType::BOOL;
    else
      CHECK(false) << "precision is not supported";
  };

  // input
  for (int i = 0; i < getNbInputs(); i++) {
    auto const& input_dims = input_desc[i].dims;

    std::vector<int> input_shape;
    for (int j = 0; j < input_dims.nbDims; j++)
      input_shape.push_back(input_dims.d[j]);

    int input_numel = 1;
    for (int k = 0; k < input_shape.size(); k++) input_numel *= input_shape[k];

    phi::DenseTensorMeta input_meta(protoType2PhiType(inputs_data_type_[i]),
                                    phi::make_ddim(input_shape));
    std::shared_ptr<phi::Allocation> input_alloc(
        new phi::Allocation((void*)(inputs[i]),  // NOLINT
                            input_numel * sizeof(int32_t),
                            place));
    (*dense_tensor_inputs)[i] =
        std::move(phi::DenseTensor(input_alloc, input_meta));
    phi_kernel_context_->EmplaceBackInput(&((*dense_tensor_inputs)[i]));
  }

  // output
  for (int i = 0; i < getNbOutputs(); i++) {
    auto const& output_dims = output_desc[i].dims;

    std::vector<int> output_shape;
    for (int j = 0; j < output_dims.nbDims; j++)
      output_shape.push_back(output_dims.d[j]);

    int output_numel = 1;
    for (int k = 0; k < output_shape.size(); k++)
      output_numel *= output_shape[k];

    phi::DenseTensorMeta output_meta(protoType2PhiType(outputs_data_type_[i]),
                                     phi::make_ddim(output_shape));
    std::shared_ptr<phi::Allocation> output_alloc(
        new phi::Allocation(reinterpret_cast<void*>(outputs[i]),
                            output_numel * sizeof(float),
                            place));
    phi::DenseTensor output_densetonsor(output_alloc, output_meta);
    (*dense_tensor_outputs)[i] =
        std::move(phi::DenseTensor(output_alloc, output_meta));
    phi_kernel_context_->EmplaceBackOutput(&((*dense_tensor_outputs)[i]));
  }

  (*phi_kernel_)(phi_kernel_context_);

  free(dense_tensor_inputs);
  free(dense_tensor_outputs);
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
