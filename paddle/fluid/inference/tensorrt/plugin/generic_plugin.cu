// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/generic_plugin.h"
#include "paddle/fluid/framework/framework.pb.h"
#include "paddle/fluid/framework/op_kernel_type.h"
#include "paddle/fluid/framework/phi_utils.h"
#include "paddle/fluid/inference/tensorrt/dynamic_shape_infermeta_registry.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/compat/op_utils.h"
#include "paddle/phi/core/kernel_context.h"
#include "paddle/phi/core/kernel_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

void BuildPhiKernelContextAttr(const framework::OpDesc& op_desc,
                               phi::KernelContext* kernel_context,
                               const phi::KernelSignature& signature,
                               const phi::Kernel& phi_kernel) {
  const phi::KernelArgsDef& args_def = phi_kernel.args_def();
  const auto& attr_names = signature.attr_names;
  const auto& attr_defs = args_def.attribute_defs();

  PADDLE_ENFORCE_EQ(
      attr_names.size(),
      attr_defs.size(),
      platform::errors::InvalidArgument(
          "The attr_names.size() should be equal to attr_defs.size()."));

  framework::AttrReader attr_reader(op_desc.GetAttrMap());

  for (size_t k = 0; k < attr_names.size(); ++k) {
    auto attr_name = attr_names[k];
    auto* attr_ptr = attr_reader.GetAttr(attr_name);
    if (attr_ptr) {
      switch (attr_defs[k].type_index) {
        case phi::AttributeType::SCALAR: {
          auto& attr = *attr_ptr;
          switch (AttrTypeID(attr)) {
            case framework::proto::AttrType::FLOAT:
              return kernel_context->EmplaceBackAttr(
                  phi::Scalar(PADDLE_GET_CONST(float, attr)));
              break;
            case framework::proto::AttrType::INT:
              return kernel_context->EmplaceBackAttr(
                  phi::Scalar(PADDLE_GET_CONST(int, attr)));
              break;
            case framework::proto::AttrType::STRING:
              return kernel_context->EmplaceBackAttr(
                  phi::Scalar(PADDLE_GET_CONST(std::string, attr)));
              break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` to Scalar when "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        } break;

        case phi::AttributeType::INT_ARRAY: {
          auto& attr = *attr_ptr;
          switch (AttrTypeID(attr)) {
            case framework::proto::AttrType::INTS:
              kernel_context->EmplaceBackAttr(std::move(
                  phi::IntArray(PADDLE_GET_CONST(std::vector<int32_t>, attr))));
              break;
            case framework::proto::AttrType::LONGS:
              kernel_context->EmplaceBackAttr(std::move(
                  phi::IntArray(PADDLE_GET_CONST(std::vector<int64_t>, attr))));
              break;
            case framework::proto::AttrType::INT:
              kernel_context->EmplaceBackAttr(
                  phi::IntArray({PADDLE_GET_CONST(int, attr)}));
              break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` to IntArray when "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        } break;

        case phi::AttributeType::SCALARS: {
          auto& attr = *attr_ptr;
          switch (AttrTypeID(attr)) {
            case framework::proto::AttrType::INTS: {
              const auto& vec = PADDLE_GET_CONST(std::vector<int32_t>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            case framework::proto::AttrType::LONGS: {
              const auto& vec = PADDLE_GET_CONST(std::vector<int64_t>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            case framework::proto::AttrType::FLOATS: {
              const auto& vec = PADDLE_GET_CONST(std::vector<float>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            case framework::proto::AttrType::FLOAT64S: {
              const auto& vec = PADDLE_GET_CONST(std::vector<double>, attr);
              std::vector<phi::Scalar> scalar_list;
              scalar_list.reserve(vec.size());
              for (const auto& val : vec) {
                scalar_list.emplace_back(val);
              }
              kernel_context->EmplaceBackAttr(std::move(scalar_list));
            } break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` to vector<Scalar> when "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        } break;

        default: {
          auto& attr = *attr_ptr;
          switch (attr_defs[k].type_index) {
            case phi::AttributeType::FLOAT32:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(float, attr));
              break;
            case phi::AttributeType::INT32:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(int, attr));
              break;
            case phi::AttributeType::BOOL:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(bool, attr));
              break;
            case phi::AttributeType::INT64:
              kernel_context->EmplaceBackAttr(PADDLE_GET_CONST(int64_t, attr));
              break;
            case phi::AttributeType::INT32S:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<int>, attr));
              break;
            case phi::AttributeType::DATA_TYPE: {
              auto data_type = paddle::framework::TransToPhiDataType(
                  static_cast<framework::proto::VarType::Type>(
                      PADDLE_GET_CONST(int, attr)));
              kernel_context->EmplaceBackAttr(data_type);
            } break;
            case phi::AttributeType::STRING:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::string, attr));
              break;
            case phi::AttributeType::INT64S:
              switch (AttrTypeID(attr)) {
                case framework::proto::AttrType::LONGS:
                  kernel_context->EmplaceBackAttr(
                      PADDLE_GET_CONST(std::vector<int64_t>, attr));
                  break;
                case framework::proto::AttrType::INTS: {
                  const auto& vector_int_attr =
                      PADDLE_GET_CONST(std::vector<int>, attr);
                  const std::vector<int64_t> vector_int64_attr(
                      vector_int_attr.begin(), vector_int_attr.end());
                  kernel_context->EmplaceBackAttr(vector_int64_attr);
                } break;
                default:
                  PADDLE_THROW(platform::errors::Unimplemented(
                      "Unsupported cast op attribute `%s` to vector<int64_t> "
                      "when ProtoAttr2PhiAttr.",
                      attr_name));
              }
              break;
            case phi::AttributeType::FLOAT32S:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<float>, attr));
              break;
            case phi::AttributeType::STRINGS:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<std::string>, attr));
              break;
            case phi::AttributeType::BOOLS:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<bool>, attr));
              break;
            case phi::AttributeType::FLOAT64S:
              kernel_context->EmplaceBackAttr(
                  PADDLE_GET_CONST(std::vector<double>, attr));
              break;
            default:
              PADDLE_THROW(platform::errors::Unimplemented(
                  "Unsupported cast op attribute `%s` when construct "
                  "ProtoAttr2PhiAttr.",
                  attr_name));
          }
        }
      }
    }
  }
  CHECK_EQ(attr_names.size(), kernel_context->AttrsSize());
}

GenericPlugin::GenericPlugin(
    const paddle::framework::proto::OpDesc& proto_op_desc,
    const InputOutPutVarInfo& in_out_info) {
  proto_op_desc_ = proto_op_desc;
  op_desc_ = std::move(framework::OpDesc(proto_op_desc_, nullptr));
  proto_op_desc_.SerializeToString(&op_meta_data_);
  inputs_data_type_ = in_out_info.inputs_data_type;
  outputs_data_type_ = in_out_info.outputs_data_type;
}

GenericPlugin::GenericPlugin(
    const paddle::framework::proto::OpDesc& proto_op_desc,
    const std::vector<int>& inputs_data_type,
    const std::vector<int>& outputs_data_type) {
  proto_op_desc_ = proto_op_desc;
  op_desc_ = std::move(framework::OpDesc(proto_op_desc_, nullptr));
  proto_op_desc_.SerializeToString(&op_meta_data_);
  inputs_data_type_ = inputs_data_type;
  outputs_data_type_ = outputs_data_type;
}

GenericPlugin::GenericPlugin(void const* serial_data, size_t serial_length) {
  DeserializeValue(&serial_data, &serial_length, &inputs_data_type_);
  DeserializeValue(&serial_data, &serial_length, &outputs_data_type_);
  std::string op_meta_data((char*)(serial_data), serial_length);  // NOLINT
  op_meta_data_ = std::move(op_meta_data);
  proto_op_desc_.ParseFromString(op_meta_data_);
  op_desc_ = std::move(framework::OpDesc(proto_op_desc_, nullptr));
}

int GenericPlugin::getNbOutputs() const TRT_NOEXCEPT {
  int res = 0;
  for (auto& i : op_desc_.Outputs()) {
    if (!i.second.empty()) res += i.second.size();
  }
  return res;
}

int GenericPlugin::getNbInputs() const TRT_NOEXCEPT {
  int res = 0;
  for (auto& i : op_desc_.Inputs()) {
    if (!i.second.empty()) res += i.second.size();
  }
  return res;
}

nvinfer1::IPluginV2DynamicExt* GenericPlugin::clone() const TRT_NOEXCEPT {
  nvinfer1::IPluginV2DynamicExt* plugin =
      new GenericPlugin(proto_op_desc_, inputs_data_type_, outputs_data_type_);
  plugin->initialize();
  return plugin;
}

void GenericPlugin::serialize(void* buffer) const TRT_NOEXCEPT {
  // inputs_data_type_
  SerializeValue(&buffer, inputs_data_type_);
  // outputs_data_type_
  SerializeValue(&buffer, outputs_data_type_);
  // serialize op_meta_data_
  std::memcpy(buffer, op_meta_data_.c_str(), op_meta_data_.size());
  reinterpret_cast<char*&>(buffer) += op_meta_data_.size();
}

bool GenericPlugin::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  if (op_desc_.Type() == "gather_nd" || op_desc_.Type() == "yolo_box") {
    if (pos == 0)
      return (in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
    if (pos == 1)
      return (in_out[pos].type == nvinfer1::DataType::kINT32) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
    if (pos == 2)
      return (in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
  } else if (op_desc_.Type() == "scatter_nd_add") {
    if (pos == 0)
      return (in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
    if (pos == 1)
      return (in_out[pos].type == nvinfer1::DataType::kINT32) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
    if (pos == 2)
      return (in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
    if (pos == 3)
      return (in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
             (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
  } else {
    return (in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
           (in_out[pos].format == nvinfer1::TensorFormat::kLINEAR);
  }
}

nvinfer1::DataType GenericPlugin::getOutputDataType(
    int index,
    const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

int GenericPlugin::initialize() TRT_NOEXCEPT {
  std::string op_type = op_desc_.Type();

  phi::KernelSignature phi_kernel_signature;
  if (phi::OpUtilsMap::Instance().HasArgumentMappingFn(op_type)) {
    const phi::ArgumentMappingFn* argument_mapping_func =
        phi::OpUtilsMap::Instance().GetArgumentMappingFn(op_type);
    PluginArgumentMappingContext argument_mapping_context(&op_desc_);
    phi_kernel_signature = (*argument_mapping_func)(argument_mapping_context);
  } else {
    phi_kernel_signature =
        phi::DefaultKernelSignatureMap::Instance().Get(op_type);
  }

  phi::KernelKey phi_kernel_key(
      phi::Backend::GPU, phi::DataLayout::ANY, phi::DataType::FLOAT32);

  PADDLE_ENFORCE_EQ(
      phi::KernelFactory::Instance().HasCompatiblePhiKernel(op_type),
      true,
      platform::errors::Fatal("%s has no compatible phi kernel!",
                              op_type.c_str()));

  const phi::Kernel& phi_kernel = phi::KernelFactory::Instance().SelectKernel(
      phi_kernel_signature.name, phi_kernel_key);
  phi_kernel_ = &phi_kernel;

  PADDLE_ENFORCE_EQ(phi_kernel_->IsValid(),
                    true,
                    platform::errors::Fatal("%s phi kernel is invalid!.",
                                            phi_kernel_signature.name));

  paddle::platform::DeviceContextPool& pool =
      paddle::platform::DeviceContextPool::Instance();
  platform::CUDAPlace place(platform::GetCurrentDeviceId());
  auto* dev_ctx = static_cast<phi::GPUContext*>(pool.Get(place));

  if (!phi_kernel_context_) {
    phi_kernel_context_ = new phi::KernelContext(dev_ctx);
    BuildPhiKernelContextAttr(
        op_desc_, phi_kernel_context_, phi_kernel_signature, phi_kernel);
  }
  if (!dense_tensor_inputs_)
    dense_tensor_inputs_ = new std::vector<phi::DenseTensor>(getNbInputs());
  if (!dense_tensor_outputs_)
    dense_tensor_outputs_ = new std::vector<phi::DenseTensor>(getNbOutputs());

  return 0;
}

nvinfer1::DimsExprs GenericPlugin::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs* inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  CHECK(output_index < getNbOutputs());
  auto& dynamic_infermeta_factory = tensorrt::DynamicMetaFnFactory::Instance();
  PADDLE_ENFORCE_EQ(dynamic_infermeta_factory.Contains(op_desc_.Type()),
                    true,
                    platform::errors::InvalidArgument(
                        "The %s op has no dynamic plugin infershape function!",
                        op_desc_.Type().c_str()));

  auto* infershape_func = dynamic_infermeta_factory.Get(op_desc_.Type());
  return infershape_func(
      output_index, inputs, nb_inputs, expr_builder, op_desc_);
}

void GenericPlugin::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* in,
    int nb_inputs,
    const nvinfer1::DynamicPluginTensorDesc* out,
    int nb_outputs) TRT_NOEXCEPT {
  CHECK(phi_kernel_context_);
  CHECK(phi_kernel_);
  CHECK(nb_inputs == getNbInputs());
  CHECK(nb_outputs == getNbOutputs());
}

// Shutdown the layer. This is called when the engine is destroyed
void GenericPlugin::terminate() TRT_NOEXCEPT {
  delete phi_kernel_context_;
  delete dense_tensor_inputs_;
  delete dense_tensor_outputs_;
}

int GenericPlugin::enqueue(const nvinfer1::PluginTensorDesc* input_desc,
                           const nvinfer1::PluginTensorDesc* output_desc,
                           const void* const* inputs,
                           void* const* outputs,
                           void* workspace,
                           hipStream_t stream) TRT_NOEXCEPT {
  platform::CUDAPlace place(platform::GetCurrentDeviceId());

  // [TODO]now generic plugin do not support FP16 and INT8 precision
  auto protoType2PhiType = [](int proto_type) -> std::pair<phi::DataType, int> {
    if (proto_type ==
        static_cast<int>(framework::proto::VarType_Type::VarType_Type_FP32))
      return {phi::DataType::FLOAT32, sizeof(float)};
    else if (proto_type ==
                 static_cast<int>(
                     framework::proto::VarType_Type::VarType_Type_INT64) ||
             proto_type ==
                 static_cast<int>(
                     framework::proto::VarType_Type::VarType_Type_INT32))
      return {phi::DataType::INT32, sizeof(int32_t)};
    else if (proto_type ==
             static_cast<int>(
                 framework::proto::VarType_Type::VarType_Type_BOOL))
      return {phi::DataType::BOOL, sizeof(bool)};
    else
      CHECK(false) << "precision is not supported";
  };

  // input
  phi_kernel_context_->ClearInputOutput();

  for (int i = 0; i < getNbInputs(); i++) {
    auto const& input_dims = input_desc[i].dims;

    std::vector<int> input_shape;
    for (int j = 0; j < input_dims.nbDims; j++)
      input_shape.push_back(input_dims.d[j]);

    int input_numel = 1;
    for (int k = 0; k < input_shape.size(); k++) input_numel *= input_shape[k];

    auto data_type_and_size = protoType2PhiType(inputs_data_type_[i]);
    phi::DenseTensorMeta input_meta(data_type_and_size.first,
                                    phi::make_ddim(input_shape));
    std::shared_ptr<phi::Allocation> input_alloc(
        new phi::Allocation((void*)(inputs[i]),  // NOLINT
                            input_numel * data_type_and_size.second,
                            place));
    (*dense_tensor_inputs_)[i] =
        std::move(phi::DenseTensor(input_alloc, input_meta));
    phi_kernel_context_->EmplaceBackInput(&((*dense_tensor_inputs_)[i]));
  }

  // output
  for (int i = 0; i < getNbOutputs(); i++) {
    auto const& output_dims = output_desc[i].dims;

    std::vector<int> output_shape;
    for (int j = 0; j < output_dims.nbDims; j++)
      output_shape.push_back(output_dims.d[j]);

    int output_numel = 1;
    for (int k = 0; k < output_shape.size(); k++)
      output_numel *= output_shape[k];

    auto data_type_and_size = protoType2PhiType(inputs_data_type_[i]);
    phi::DenseTensorMeta output_meta(data_type_and_size.first,
                                     phi::make_ddim(output_shape));
    std::shared_ptr<phi::Allocation> output_alloc(
        new phi::Allocation(reinterpret_cast<void*>(outputs[i]),
                            output_numel * data_type_and_size.second,
                            place));
    phi::DenseTensor output_densetonsor(output_alloc, output_meta);
    (*dense_tensor_outputs_)[i] =
        std::move(phi::DenseTensor(output_alloc, output_meta));
    phi_kernel_context_->EmplaceBackOutput(&((*dense_tensor_outputs_)[i]));
  }

  CHECK_EQ(phi_kernel_context_->InputsSize(), getNbInputs());
  CHECK_EQ(phi_kernel_context_->OutputsSize(), getNbOutputs());

  (*phi_kernel_)(phi_kernel_context_);

  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
