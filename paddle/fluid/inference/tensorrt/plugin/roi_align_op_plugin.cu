#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "paddle/fluid/inference/tensorrt/plugin/roi_align_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

template <class T>
__inline__ __device__ T BilinearInterpolate(const T* input_data,
                                            const int height, const int width,
                                            T y, T x) {
  if (y < -1.f || y > height || x < -1.f || x > width) return 0;
  y = y <= 0.f ? 0.f : y;
  x = x <= 0.f ? 0.f : x;
  int y_low = static_cast<int>(y);
  int x_low = static_cast<int>(x);
  int y_high;
  int x_high;
  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = static_cast<T>(y_low);
  } else {
    y_high = y_low + 1;
  }
  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = static_cast<T>(x_low);
  } else {
    x_high = x_low + 1;
  }
  T ly = y - y_low, lx = x - x_low;
  T hy = 1.f - ly, hx = 1.f - lx;
  T v1 = input_data[y_low * width + x_low];
  T v2 = input_data[y_low * width + x_high];
  T v3 = input_data[y_high * width + x_low];
  T v4 = input_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename T, typename OutT, bool USE_SMEM>
__global__ void GPUROIAlignOpt(const int nthreads,
                               const T* __restrict__ input_data,
                               const T* __restrict__ input_rois,
                               const float spatial_scale, const int channels,
                               const int height, const int width,
                               const int pooled_height, const int pooled_width,
                               const int sampling_ratio, const int num_rois,
                               OutT* __restrict__ output_data) {
  const int batch = blockIdx.x;
  const int channel = blockIdx.y;
  const T* offset_input_data =
      input_data + (batch * channels + channel) * height * width;
  extern __shared__ T s_input_data[];
  if (USE_SMEM) {
    for (int idx = threadIdx.x; idx < height * width; idx += blockDim.x) {
      s_input_data[idx] = offset_input_data[idx];
    }
    __syncthreads();
  }
  for (int idx = threadIdx.x; idx < num_rois * pooled_height * pooled_width;
       idx += blockDim.x) {
    const int pw = idx % pooled_width;
    const int ph = (idx / pooled_width) % pooled_height;
    const int roi_idx = (idx / pooled_width / pooled_height) % num_rois;
    const int n = batch * num_rois + roi_idx;
    const float4 rois_offset = reinterpret_cast<const float4*>(input_rois)[n];
    const T roi_xmin = rois_offset.x * spatial_scale;
    const T roi_ymin = rois_offset.y * spatial_scale;
    const T roi_xmax = rois_offset.z * spatial_scale;
    const T roi_ymax = rois_offset.w * spatial_scale;
    const T roi_width = max(roi_xmax - roi_xmin, static_cast<T>(1.f));
    const T roi_height = max(roi_ymax - roi_ymin, static_cast<T>(1.f));
    const T bin_size_h = roi_height / static_cast<T>(pooled_height);
    const T bin_size_w = roi_width / static_cast<T>(pooled_width);
    const int roi_bin_grid_h = (sampling_ratio > 0)
                                   ? sampling_ratio
                                   : ceil(roi_height / pooled_height);
    const int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);
    const T count = roi_bin_grid_h * roi_bin_grid_w;

    T output_val = 0.f;
    for (int iy = 0; iy < roi_bin_grid_h; ++iy) {
      const T y = roi_ymin + ph * bin_size_h +
                  static_cast<T>(iy + .5f) * bin_size_h /
                      static_cast<T>(roi_bin_grid_h);
      for (int ix = 0; ix < roi_bin_grid_w; ++ix) {
        const T x = roi_xmin + pw * bin_size_w +
                    static_cast<T>(ix + .5f) * bin_size_w /
                        static_cast<T>(roi_bin_grid_w);
        if (USE_SMEM) {
          T val = BilinearInterpolate<T>(s_input_data, height, width, y, x);
          output_val += val;
        } else {
          T val =
              BilinearInterpolate<T>(offset_input_data, height, width, y, x);
          output_val += val;
        }
      }
    }
    output_val /= count;
    const int out_offset =
        batch * num_rois * channels * pooled_height * pooled_width +
        roi_idx * channels * pooled_height * pooled_width +
        channel * pooled_height * pooled_width + ph * pooled_width + pw;
    output_data[out_offset] = static_cast<OutT>(output_val);
  }
}

#if IS_TRT_VERSION_GE(6000)
RoiAlignPluginDynamic::RoiAlignPluginDynamic(const nvinfer1::DataType data_type,
                                             const int pooled_height,
                                             const int pooled_width,
                                             float spatial_scale,
                                             int sampling_ratio)
    : data_type_(data_type),
      pooled_height_(pooled_height),
      pooled_width_(pooled_width),
      spatial_scale_(spatial_scale),
      sampling_ratio_(sampling_ratio) {
  bool data_type_is_valid = data_type_ == nvinfer1::DataType::kFLOAT ||
                            data_type_ == nvinfer1::DataType::kHALF;
  PADDLE_ENFORCE_EQ(data_type_is_valid, true,
                    platform::errors::InvalidArgument(
                        "TRT RoiAlign plugin only accepts kFLOAT(%d) or "
                        "kHALF(%d) data type, but the received data type = %d",
                        static_cast<int>(nvinfer1::DataType::kFLOAT),
                        static_cast<int>(nvinfer1::DataType::kHALF),
                        static_cast<int>(data_type_)));

  PADDLE_ENFORCE_GT(pooled_height_, 0,
                    platform::errors::InvalidArgument(
                        "TRT RoiAlign plugin only accepts pooled_height "
                        "greater than %d, but the received pooled_height = %d",
                        0, pooled_height_));

  PADDLE_ENFORCE_GT(pooled_width_, 0,
                    platform::errors::InvalidArgument(
                        "TRT RoiAlign plugin only accepts pooled_width greater "
                        "than %d, but the received pooled_width = %d",
                        0, pooled_height_));

  PADDLE_ENFORCE_GT(spatial_scale_, 0.f,
                    platform::errors::InvalidArgument(
                        "TRT RoiAlign plugin only accepts spatial_scale "
                        "greater than %f, but the received spatial_scale = %f",
                        0, spatial_scale_));

  int smem_per_block = -1;
  int device = -1;
  hipGetDevice(&device);

  PADDLE_ENFORCE_GE(
      device, 0,
      platform::errors::InvalidArgument(
          "The cuda device ID should be greater than %d, but device ID is %d",
          0, device));

  hipDeviceGetAttribute(&smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock,
                         device);
  smem_per_block_ = smem_per_block;
}

RoiAlignPluginDynamic::RoiAlignPluginDynamic(void const* data, size_t length) {
  DeserializeValue(&data, &length, &data_type_);
  DeserializeValue(&data, &length, &pooled_height_);
  DeserializeValue(&data, &length, &pooled_width_);
  DeserializeValue(&data, &length, &spatial_scale_);
  DeserializeValue(&data, &length, &sampling_ratio_);
  int smem_per_block = -1;
  int device = -1;
  hipGetDevice(&device);
  PADDLE_ENFORCE_GE(
      device, 0,
      platform::errors::InvalidArgument(
          "The cuda device ID should be greater than %d, but device ID is %d",
          0, device));
  hipDeviceGetAttribute(&smem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock,
                         device);
  smem_per_block_ = smem_per_block;
}

nvinfer1::IPluginV2DynamicExt* RoiAlignPluginDynamic::clone() const {
  auto* plugin =
      new RoiAlignPluginDynamic(data_type_, pooled_height_, pooled_width_,
                                spatial_scale_, sampling_ratio_);
  plugin->setPluginNamespace(namespace_.c_str());
  return plugin;
}

nvinfer1::DimsExprs RoiAlignPluginDynamic::getOutputDimensions(
    int outputIndex, const nvinfer1::DimsExprs* inputs, int nbInputs,
    nvinfer1::IExprBuilder& exprBuilder) {
  nvinfer1::DimsExprs ret{};
  ret.nbDims = 4;
  ret.d[0] = inputs[1].d[0];  // roi
  ret.d[1] = inputs[0].d[1];  // X
  ret.d[2] = exprBuilder.constant(pooled_height_);
  ret.d[3] = exprBuilder.constant(pooled_width_);
  return ret;
}

bool RoiAlignPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* inOut, int nbInputs,
    int nbOutputs) {
  if (inOut[pos].format != nvinfer1::TensorFormat::kLINEAR) {
    return false;
  }
  if (pos < 2) {  // input
    return inOut[pos].type == nvinfer1::DataType::kFLOAT;
  }
  return inOut[pos].type == data_type_;
}

void RoiAlignPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs) {}

size_t RoiAlignPluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const {
  return 0;
}

template <typename T, typename OutT>
int RoiAlignPluginDynamic::enqueue_impl(
    const nvinfer1::PluginTensorDesc* inputDesc,
    const nvinfer1::PluginTensorDesc* outputDesc, const void* const* inputs,
    void* const* outputs, void* workspace, hipStream_t stream) {
  auto in_dims = inputDesc[0].dims;
  auto rois_dims = inputDesc[1].dims;
  auto out_dims = outputDesc[0].dims;

  int rois_num = rois_dims.d[0];
  if (rois_num == 0) return hipGetLastError() != hipSuccess;

  int batch = in_dims.d[0];
  int channels = in_dims.d[1];
  int height = in_dims.d[2];
  int width = in_dims.d[3];

  int output_size =
      out_dims.d[0] * out_dims.d[1] * out_dims.d[2] * out_dims.d[3];

  const dim3 blocks(batch, channels);
  const int threads = 512;

  if (smem_per_block_ < width * height * sizeof(T)) {
    GPUROIAlignOpt<T, OutT, false><<<blocks, threads, 0, stream>>>(
        output_size, static_cast<const T*>(inputs[0]),
        static_cast<const T*>(inputs[1]), spatial_scale_, channels, height,
        width, pooled_height_, pooled_width_, sampling_ratio_, rois_num / batch,
        static_cast<OutT*>(outputs[0]));
  } else {
    GPUROIAlignOpt<
        T, OutT, true><<<blocks, threads, width * height * sizeof(T), stream>>>(
        output_size, static_cast<const T*>(inputs[0]),
        static_cast<const T*>(inputs[1]), spatial_scale_, channels, height,
        width, pooled_height_, pooled_width_, sampling_ratio_, rois_num / batch,
        static_cast<OutT*>(outputs[0]));
  }

  return hipGetLastError() != hipSuccess;
}

int RoiAlignPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
                                   const nvinfer1::PluginTensorDesc* outputDesc,
                                   const void* const* inputs,
                                   void* const* outputs, void* workspace,
                                   hipStream_t stream) {
  PADDLE_ENFORCE_EQ(outputDesc[0].type, data_type_,
                    platform::errors::InvalidArgument(
                        "TRT RoiAlignPluginDynamic expects outputDesc[0].type "
                        "equal to data_type_"));

  if (data_type_ == nvinfer1::DataType::kHALF) {
    return enqueue_impl<float, half>(inputDesc, outputDesc, inputs, outputs,
                                     workspace, stream);
  }
  return enqueue_impl<float, float>(inputDesc, outputDesc, inputs, outputs,
                                    workspace, stream);
}

nvinfer1::DataType RoiAlignPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const {
  return data_type_;
}

const char* RoiAlignPluginDynamic::getPluginType() const {
  return "roi_align_plugin_dynamic";
}

int RoiAlignPluginDynamic::getNbOutputs() const { return 1; }

int RoiAlignPluginDynamic::initialize() { return 0; }

void RoiAlignPluginDynamic::terminate() {}

size_t RoiAlignPluginDynamic::getSerializationSize() const {
  size_t serialize_size = 0;
  serialize_size += SerializedSize(data_type_);
  serialize_size += SerializedSize(pooled_height_);
  serialize_size += SerializedSize(pooled_width_);
  serialize_size += SerializedSize(spatial_scale_);
  serialize_size += SerializedSize(sampling_ratio_);
  return serialize_size;
}

void RoiAlignPluginDynamic::serialize(void* buffer) const {
  SerializeValue(&buffer, data_type_);
  SerializeValue(&buffer, pooled_height_);
  SerializeValue(&buffer, pooled_width_);
  SerializeValue(&buffer, spatial_scale_);
  SerializeValue(&buffer, sampling_ratio_);
}

void RoiAlignPluginDynamic::destroy() {}

RoiAlignPluginDynamicCreator::RoiAlignPluginDynamicCreator() {}

void RoiAlignPluginDynamicCreator::setPluginNamespace(
    const char* lib_namespace) {
  namespace_ = std::string(lib_namespace);
}

const char* RoiAlignPluginDynamicCreator::getPluginNamespace() const {
  return namespace_.c_str();
}

const char* RoiAlignPluginDynamicCreator::getPluginName() const {
  return "roi_align_plugin_dynamic";
}

const char* RoiAlignPluginDynamicCreator::getPluginVersion() const {
  return "1";
}

const nvinfer1::PluginFieldCollection*
RoiAlignPluginDynamicCreator::getFieldNames() {
  return &field_collection_;
}

nvinfer1::IPluginV2Ext* RoiAlignPluginDynamicCreator::createPlugin(
    const char* name, const nvinfer1::PluginFieldCollection* fc) {
  const nvinfer1::PluginField* fields = fc->fields;
  return nullptr;
}

nvinfer1::IPluginV2Ext* RoiAlignPluginDynamicCreator::deserializePlugin(
    const char* name, const void* serial_data, size_t serial_length) {
  auto plugin = new RoiAlignPluginDynamic(serial_data, serial_length);
  plugin->setPluginNamespace(namespace_.c_str());
  return plugin;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
