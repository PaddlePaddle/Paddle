// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/layer_norm_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/operators/layer_norm_op.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

LayerNormPlugin *CreateLayerNormPluginDeserialize(const void *buffer,
                                                  size_t length) {
  return new LayerNormPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("layer_norm_plugin", CreateLayerNormPluginDeserialize);

int LayerNormPlugin::initialize() {
  // hipMalloc(&p_gpu_bias_, sizeof(float) * bias_.size());
  // hipMemcpy(p_gpu_bias_, bias_.data(), bias_.size() * sizeof(float),
  //            hipMemcpyHostToDevice);
  // hipMalloc(&p_gpu_scale_, sizeof(float) * scale_.size());
  // hipMemcpy(p_gpu_scale_, scale_.data(), scale_.size() * sizeof(float),
  //            hipMemcpyHostToDevice);
  return 0;
}

nvinfer1::Dims LayerNormPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *inputDims, int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

int LayerNormPlugin::enqueue(int batch_size, const void *const *inputs,
                             void **outputs, void *workspace,
                             hipStream_t stream) {
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  float *output = reinterpret_cast<float **>(outputs)[0];
  int begin_norm_axis = begin_norm_axis_;
  float eps = eps_;

  int n = batch_size;
  int c = input_dims.d[begin_norm_axis - 1];

  scale_t.Resize(framework::make_ddim({n, c}));
  bias_t.Resize(framework::make_ddim({n, c}));
  int device_id;
  hipGetDevice(&device_id);
  float *scale_d = scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));

  for (int i = 0; i < n; i++) {
    hipMemcpyAsync(scale_d + i * c, scale_.data(), sizeof(float) * c,
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(bias_d + i * c, bias_.data(), sizeof(float) * c,
                    hipMemcpyHostToDevice, stream);
  }

  // const float *bias = p_gpu_bias_;
  // const float *scale = p_gpu_scale_;

  std::vector<int> input_shape;
  input_shape.push_back(batch_size);
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }

  paddle::operators::LayerNormDirectCUDAFunctor<float> layer_norm;
  layer_norm(stream, input, input_shape, bias_d, scale_d, output, mean_shape_,
             variance_shape_, begin_norm_axis, eps);

  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
