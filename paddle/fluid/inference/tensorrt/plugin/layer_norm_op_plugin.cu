// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/layer_norm_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/operators/layer_norm_op.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

LayerNormPlugin *CreateLayerNormPluginDeserialize(const void *buffer,
                                                  size_t length) {
  return new LayerNormPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("layer_norm_plugin", CreateLayerNormPluginDeserialize);

int LayerNormPlugin::initialize() { return 0; }

nvinfer1::Dims LayerNormPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *inputDims, int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

int LayerNormPlugin::enqueue(int batch_size, const void *const *inputs,
                             void **outputs, void *workspace,
                             hipStream_t stream) {
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  float *output = reinterpret_cast<float **>(outputs)[0];
  int begin_norm_axis = begin_norm_axis_;
  float eps = eps_;

  std::vector<int> input_shape;
  input_shape.push_back(batch_size);
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }
  const auto input_ddim = framework::make_ddim(input_shape);
  auto matrix_dim = framework::flatten_to_2d(input_ddim, begin_norm_axis);
  int feature_size = static_cast<int>(matrix_dim[1]);
  PADDLE_ENFORCE_EQ(feature_size, scale_.size(),
                    platform::errors::InvalidArgument(
                        "scale's size should be equal to the feature_size,"
                        "but got feature_size:%d, scale's size:%d.",
                        feature_size, scale_.size()));
  PADDLE_ENFORCE_EQ(feature_size, bias_.size(),
                    platform::errors::InvalidArgument(
                        "bias's size should be equal to the feature_size,"
                        "but got feature_size:%d, bias's size:%d.",
                        feature_size, bias_.size()));

  scale_t.Resize(framework::make_ddim({feature_size}));
  bias_t.Resize(framework::make_ddim({feature_size}));
  mean_t.Resize(framework::make_ddim(mean_shape_));
  variance_t.Resize(framework::make_ddim(variance_shape_));
  int device_id;
  hipGetDevice(&device_id);
  float *scale_d = scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *mean_d = mean_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *variance_d =
      variance_t.mutable_data<float>(platform::CUDAPlace(device_id));
  hipMemcpyAsync(scale_d, scale_.data(), sizeof(float) * feature_size,
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(bias_d, bias_.data(), sizeof(float) * feature_size,
                  hipMemcpyHostToDevice, stream);

  paddle::operators::LayerNormDirectCUDAFunctor<float> layer_norm;
  layer_norm(stream, input, input_shape, bias_d, scale_d, output, mean_d,
             variance_d, begin_norm_axis, eps);
  return hipGetLastError() != hipSuccess;
}

nvinfer1::DimsExprs LayerNormPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputDims, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  return inputDims[0];
}

bool LayerNormPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of layernorm plugin shoule not be nullptr."));
  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    // TODO(Shangzhizhou) FP16 support
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType LayerNormPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0,
                    platform::errors::InvalidArgument(
                        "The LayerNormPlugin only has one input, so the "
                        "index value should be 0, but get %d.",
                        index));
  return input_types[0];
}

int LayerNormPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) {
  const auto &input_dims = input_desc[0].dims;
  int begin_norm_axis = begin_norm_axis_;
  float eps = eps_;

  std::vector<int> input_shape;
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }
  const auto input_ddim = framework::make_ddim(input_shape);
  auto matrix_dim = framework::flatten_to_2d(input_ddim, begin_norm_axis);
  int feature_size = static_cast<int>(matrix_dim[1]);
  PADDLE_ENFORCE_EQ(feature_size, scale_.size(),
                    platform::errors::InvalidArgument(
                        "scale's size should be equal to the feature_size,"
                        "but got feature_size:%d, scale's size:%d.",
                        feature_size, scale_.size()));
  PADDLE_ENFORCE_EQ(feature_size, bias_.size(),
                    platform::errors::InvalidArgument(
                        "bias's size should be equal to the feature_size,"
                        "but got feature_size:%d, bias's size:%d.",
                        feature_size, bias_.size()));
  int device_id;
  hipGetDevice(&device_id);
  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. LayerNorm-->fp32";
    const float *input = reinterpret_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    scale_t.Resize(framework::make_ddim({feature_size}));
    bias_t.Resize(framework::make_ddim({feature_size}));
    mean_t.Resize(framework::make_ddim(mean_shape_));
    variance_t.Resize(framework::make_ddim(variance_shape_));

    float *scale_d =
        scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
    float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));
    float *mean_d = mean_t.mutable_data<float>(platform::CUDAPlace(device_id));
    float *variance_d =
        variance_t.mutable_data<float>(platform::CUDAPlace(device_id));

    hipMemcpyAsync(scale_d, scale_.data(), sizeof(float) * feature_size,
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(bias_d, bias_.data(), sizeof(float) * feature_size,
                    hipMemcpyHostToDevice, stream);

    paddle::operators::LayerNormDirectCUDAFunctor<float> layer_norm;
    layer_norm(stream, input, input_shape, bias_d, scale_d, output, mean_d,
               variance_d, begin_norm_axis, eps);
  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. LayerNorm-->fp16";
    const half *input = reinterpret_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    size_t mean_shape_product = 1;
    for (auto s : mean_shape_) {
      mean_shape_product *= s;
    }
    size_t variance_shape_product = 1;
    for (auto s : variance_shape_) {
      variance_shape_product *= s;
    }
    if (!scale_gpu_half_d_) {
      hipMalloc(&scale_gpu_half_d_, feature_size * sizeof(half));
    }
    if (!bias_gpu_half_d_) {
      hipMalloc(&bias_gpu_half_d_, feature_size * sizeof(half));
    }
    if (!mean_gpu_half_d_) {
      hipMalloc(&mean_gpu_half_d_, mean_shape_product * sizeof(half));
    }
    if (!variance_gpu_half_d_) {
      hipMalloc(&variance_gpu_half_d_, variance_shape_product * sizeof(half));
    }

    half *scale_cpu_half =
        static_cast<half *>(malloc(feature_size * sizeof(half)));
    half *bias_cpu_half =
        static_cast<half *>(malloc(feature_size * sizeof(half)));
    PADDLE_ENFORCE_EQ(
        scale_cpu_half && bias_cpu_half, true,
        platform::errors::Unavailable("Out of memory, malloc size %d.",
                                      feature_size * sizeof(half)));

    for (int i = 0; i < feature_size; i++) {
      scale_cpu_half[i] = static_cast<half>(scale_[i]);
      bias_cpu_half[i] = static_cast<half>(bias_[i]);
    }
    hipMemcpyAsync(scale_gpu_half_d_, scale_cpu_half,
                    sizeof(half) * feature_size, hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(bias_gpu_half_d_, bias_cpu_half,
                    sizeof(half) * feature_size, hipMemcpyHostToDevice,
                    stream);
    free(scale_cpu_half);
    free(bias_cpu_half);

    paddle::operators::LayerNormDirectCUDAFunctor<half> layer_norm;
    layer_norm(stream, input, input_shape, bias_gpu_half_d_, scale_gpu_half_d_,
               output, mean_gpu_half_d_, variance_gpu_half_d_, begin_norm_axis,
               eps);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The layer_norm tensorRT plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The LayerNorm TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
