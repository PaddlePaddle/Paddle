// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/layer_norm_op_plugin.h"
#include "paddle/fluid/operators/layer_norm_op.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

int LayerNormPlugin::initialize() TRT_NOEXCEPT { return 0; }

nvinfer1::Dims LayerNormPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *inputDims, int nbInputs) TRT_NOEXCEPT {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

int LayerNormPlugin::enqueue(int batch_size, const void *const *inputs,
#if IS_TRT_VERSION_LT(8000)
                             void **outputs, void *workspace,
#else
                             void *const *outputs, void *workspace,
#endif
                             hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  float *output = reinterpret_cast<float *const *>(outputs)[0];
  int begin_norm_axis = begin_norm_axis_;
  float eps = eps_;

  std::vector<int> input_shape;
  input_shape.push_back(batch_size);
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }
  const auto input_ddim = phi::make_ddim(input_shape);
  auto matrix_dim = phi::flatten_to_2d(input_ddim, begin_norm_axis);
  int feature_size = static_cast<int>(matrix_dim[1]);
  PADDLE_ENFORCE_EQ(feature_size, scale_.size(),
                    platform::errors::InvalidArgument(
                        "scale's size should be equal to the feature_size,"
                        "but got feature_size:%d, scale's size:%d.",
                        feature_size, scale_.size()));
  PADDLE_ENFORCE_EQ(feature_size, bias_.size(),
                    platform::errors::InvalidArgument(
                        "bias's size should be equal to the feature_size,"
                        "but got feature_size:%d, bias's size:%d.",
                        feature_size, bias_.size()));

  scale_t.Resize(phi::make_ddim({feature_size}));
  bias_t.Resize(phi::make_ddim({feature_size}));
  mean_t.Resize(phi::make_ddim(mean_shape_));
  variance_t.Resize(phi::make_ddim(variance_shape_));
  int device_id;
  hipGetDevice(&device_id);
  float *scale_d = scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *mean_d = mean_t.mutable_data<float>(platform::CUDAPlace(device_id));
  float *variance_d =
      variance_t.mutable_data<float>(platform::CUDAPlace(device_id));
  hipMemcpyAsync(scale_d, scale_.data(), sizeof(float) * feature_size,
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(bias_d, bias_.data(), sizeof(float) * feature_size,
                  hipMemcpyHostToDevice, stream);

  paddle::operators::LayerNormDirectCUDAFunctor<float> layer_norm;
  layer_norm(stream, input, input_shape, bias_d, scale_d, output, mean_d,
             variance_d, begin_norm_axis, eps);
  return hipGetLastError() != hipSuccess;
}

nvinfer1::DimsExprs LayerNormPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputDims, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  return inputDims[0];
}

bool LayerNormPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of layernorm plugin shoule not be nullptr."));
  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    // TODO(Shangzhizhou) FP16 support
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType LayerNormPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(index, 0,
                    platform::errors::InvalidArgument(
                        "The LayerNormPlugin only has one input, so the "
                        "index value should be 0, but get %d.",
                        index));
  return input_types[0];
}

int LayerNormPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = input_desc[0].dims;
  int begin_norm_axis = begin_norm_axis_;
  float eps = eps_;

  std::vector<int> input_shape;
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }
  const auto input_ddim = phi::make_ddim(input_shape);
  auto matrix_dim = phi::flatten_to_2d(input_ddim, begin_norm_axis);
  int feature_size = static_cast<int>(matrix_dim[1]);
  PADDLE_ENFORCE_EQ(feature_size, scale_.size(),
                    platform::errors::InvalidArgument(
                        "scale's size should be equal to the feature_size,"
                        "but got feature_size:%d, scale's size:%d.",
                        feature_size, scale_.size()));
  PADDLE_ENFORCE_EQ(feature_size, bias_.size(),
                    platform::errors::InvalidArgument(
                        "bias's size should be equal to the feature_size,"
                        "but got feature_size:%d, bias's size:%d.",
                        feature_size, bias_.size()));
  int device_id;
  hipGetDevice(&device_id);
  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. LayerNorm-->fp32";
    const float *input = reinterpret_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    scale_t.Resize(phi::make_ddim({feature_size}));
    bias_t.Resize(phi::make_ddim({feature_size}));
    mean_t.Resize(phi::make_ddim(mean_shape_));
    variance_t.Resize(phi::make_ddim(variance_shape_));

    float *scale_d =
        scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
    float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));
    float *mean_d = mean_t.mutable_data<float>(platform::CUDAPlace(device_id));
    float *variance_d =
        variance_t.mutable_data<float>(platform::CUDAPlace(device_id));

    hipMemcpyAsync(scale_d, scale_.data(), sizeof(float) * feature_size,
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(bias_d, bias_.data(), sizeof(float) * feature_size,
                    hipMemcpyHostToDevice, stream);

    paddle::operators::LayerNormDirectCUDAFunctor<float> layer_norm;
    layer_norm(stream, input, input_shape, bias_d, scale_d, output, mean_d,
               variance_d, begin_norm_axis, eps);
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The LayerNorm TRT Plugin's input type should be float."));
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
