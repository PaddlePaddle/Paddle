/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/inference/tensorrt/plugin/group_norm_op_plugin.h"
#include "paddle/phi/kernels/group_norm_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
using DataLayout = phi::DataLayout;

int GroupNormPlugin::initialize() TRT_NOEXCEPT { 
  
  if(!with_fp16_){
  // if use fp32
    hipMalloc(&scale_gpu_, sizeof(float)*scale_.size())
    hipMalloc(&bias_gpu_, sizeof(float)*bias_.size())
    hipMemcpy(scale_gpu_,
               scale_.date(),
               scale_.size()*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_.date(),
               bias_.size()*sizeof(float),
               hipMemcpyHostToDevice)
  } else {
  // if use fp16
    std::vector<half> scale_half(scale_.size());
    std::vector<half> bias_half(bias_.size());
    for(i=0;i<scale_.size();++i){
      scale_half[i]=static_cast<half>(scale_[i]);
    }
    for(i=0;i<bias_.size();++i){
      bias_half[i]=static_cast<half>(bias_[i]);
    }
    hipMalloc(&scale_gpu_, sizeof(half)*scale_.size())
    hipMalloc(&bias_gpu_, sizeof(half)*bias_.size())
    hipMemcpy(scale_gpu_,
               scale_half.date(),
               scale_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_half.date(),
               bias_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
  }
  return 0; 
}

bool GroupNormPlugin::supportsFormat(
    nvinfer1::DataType type, nvinfer1::PluginFormat format) const TRT_NOEXCEPT {
  if (with_fp16_) {
    return ((type == nvinfer1::DataType::kFLOAT ||
             type == nvinfer1::DataType::kHALF) &&
            (format == nvinfer1::PluginFormat::kLINEAR));
  } else {
    return ((type == nvinfer1::DataType::kFLOAT) &&
            (format == nvinfer1::PluginFormat::kLINEAR));
  }
}

nvinfer1::Dims GroupNormPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *inputDims, int nbInputs) TRT_NOEXCEPT {
  return inputDims[0];
}

int GroupNormPlugin::enqueue(int batch_size,
                             const void *const *inputs,
#if IS_TRT_VERSION_LT(8000)
                             void **outputs,
                             void *workspace,
#else
                             void *const *outputs,
                             void *workspace,
#endif
                             hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = this->getInputDims(0);
  int groups = groups_;
  float eps = eps_;
  std::vector<int> input_shape;
  input_shape.push_back(batch_size);
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }
  const auto input_ddim = phi::make_ddim(input_shape);

  int C = input_shape[1];

  PADDLE_ENFORCE_EQ(
      C,
      scale_.size(),
      platform::errors::InvalidArgument(
          "scale's size should be equal to the channel number in groupnorm,"
          "but got channel number:%d, scale's size:%d.",
          C,
          scale_.size()));
  PADDLE_ENFORCE_EQ(
      C,
      bias_.size(),
      platform::errors::InvalidArgument(
          "bias's size should be equal to the channel number in groupnorm,"
          "but got channel number:%d, bias's size:%d.",
          C,
          bias_.size()));

  int device_id;
  hipGetDevice(&device_id);
  mean_t.Resize(phi::make_ddim(batched_mean_shape));
  variance_t.Resize(phi::make_ddim(batched_variance_shape));
  float *mean_d = mean_t.mutable_data<float>(platform::CUDAPlace(device_id));
  phi::DenseTensor temp_variance_t;
  temp_variance_t.Resize(phi::make_ddim(variance_shape_));
  float *temp_variance_d =
      temp_variance_t.mutable_data<float>(platform::CUDAPlace(device_id));
  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT){
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp32";
    const float *input = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<float> group_norm;
    group_norm(stream,
             input,
             input_shape,
             bias_gpu_,
             scale_gpu_,
             temp_variance_d,
             groups_,
             eps_,
             output,
             mean_d,
             variance_d,
             DataLayout::kNCHW);
  } else if (input_type == nvinfer1::DataType::kHALF){
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp16";
    const half *input = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<half, float> group_norm;
    group_norm(stream,
             input,
             input_shape,
             bias_gpu_,
             scale_gpu_,
             temp_variance_d,
             groups_,
             eps_,
             output,
             mean_d,
             variance_d,
             DataLayout::kNCHW);
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The GroupNorm TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
nvinfer1::DimsExprs GroupNormPluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputDims,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  return inputDims[0];
}

bool GroupNormPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of groupnorm plugin shoule not be nullptr."));
  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));
  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
      return ((in.type == nvinfer1::DataType::kFLOAT ||
               in.type == nvinfer1::DataType::kHALF) &&
              (in.format == nvinfer1::PluginFormat::kLINEAR));
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType GroupNormPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(index,
                    0,
                    platform::errors::InvalidArgument(
                        "The groupnorm Plugin only has one input, so the "
                        "index value should be 0, but get %d.",
                        index));
  return input_types[0];
}
int GroupNormPluginDynamic::initialize() TRT_NOEXCEPT {
if(!with_fp16_){
  // if use fp32
    hipMalloc(&scale_gpu_, sizeof(float)*scale_.size())
    hipMalloc(&bias_gpu_, sizeof(float)*bias_.size())
    hipMemcpy(scale_gpu_,
               scale_.date(),
               scale_.size()*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_.date(),
               bias_.size()*sizeof(float),
               hipMemcpyHostToDevice)
  } else {
  // if use fp16
    std::vector<half> scale_half(scale_.size());
    std::vector<half> bias_half(bias_.size());
    for(i=0;i<scale_.size();++i){
      scale_half[i]=static_cast<half>(scale_[i]);
    }
    for(i=0;i<bias_.size();++i){
      bias_half[i]=static_cast<half>(bias_[i]);
    }
    hipMalloc(&scale_gpu_, sizeof(half)*scale_.size())
    hipMalloc(&bias_gpu_, sizeof(half)*bias_.size())
    hipMemcpy(scale_gpu_,
               scale_half.date(),
               scale_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_half.date(),
               bias_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
  }
  return 0;
}

int GroupNormPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = input_desc[0].dims;
  int groups = groups_;
  float eps = eps_;

  std::vector<int> input_shape;
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }

  const auto input_ddim = phi::make_ddim(input_shape);

  int C = input_shape[1];
  int image_size = input_shape[2] * input_shape[3];
  int batchSize = input_shape[0];
  std::vector<int64_t> batched_mean_shape = {batchSize * mean_shape_[0]};
  std::vector<int64_t> batched_variance_shape = {batchSize *
                                                 variance_shape_[0]};
  PADDLE_ENFORCE_EQ(
      C,
      scale_.size(),
      platform::errors::InvalidArgument(
          "scale's size should be equal to the channel number in groupnorm,"
          "but got feature_size:%d, scale's size:%d.",
          C,
          scale_.size()));
  PADDLE_ENFORCE_EQ(
      C,
      bias_.size(),
      platform::errors::InvalidArgument(
          "bias's size should be equal to the channel number in groupnorm,"
          "but got feature_size:%d, bias's size:%d.",
          C,
          bias_.size()));

  int device_id;
  hipGetDevice(&device_id);
  auto input_type = input_desc[0].type;
  variance_t.Resize(phi::make_ddim(batched_variance_shape));
  float *variance_d =
      variance_t.mutable_data<float>(platform::CUDAPlace(device_id));
  phi::DenseTensor temp_variance_t;
  temp_variance_t.Resize(phi::make_ddim(batched_variance_shape));
  float *temp_variance_d =
      temp_variance_t.mutable_data<float>(platform::CUDAPlace(device_id));

  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp32";
    const float *input = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<float> group_norm;
    group_norm(stream,
               input,
               input_shape,
               bias_d,
               scale_d,
               temp_variance_d,
               groups,
               eps,
               output,
               mean_d,
               variance_d,
               DataLayout::kNCHW);
  } else if (input_type == nvinfer1::DataType::kHALF) {
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp16";
    const half *input = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<half,float> group_norm;
    group_norm(stream,
               input,
               input_shape,
               bias_d,
               scale_d,
               temp_variance_d,
               groups,
               eps,
               output,
               mean_d,
               variance_d,
               DataLayout::kNCHW);
  } else {
    // input not float
    PADDLE_THROW(platform::errors::Fatal(
        "The Groupnorm TRT Plugin's only support fp32 input"));
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
