#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/inference/tensorrt/plugin/group_norm_op_plugin.h"
#include "paddle/phi/kernels/group_norm_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
using DataLayout = phi::DataLayout;

// TODO wangbojun for debug
template<typename T>
__global__ void print_float(const T *src, int start_index, int end_index, int numPerRow=49, int stride=1){
  printf("start print float \r\n");
  for (int i=start_index;i<end_index;i+=stride){
    printf("%.5e, ",static_cast<double>(src[i]));
    if((i-start_index)/stride%numPerRow==numPerRow-1){
      printf("\r\n");
    }
  }
}

int GroupNormPlugin::initialize() TRT_NOEXCEPT { 
  if(!with_fp16_){
  // if use fp32
    hipMalloc(&scale_gpu_, sizeof(float)*scale_.size());
    hipMalloc(&bias_gpu_, sizeof(float)*bias_.size());
    hipMemcpy(scale_gpu_,
               scale_.data(),
               scale_.size()*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_.data(),
               bias_.size()*sizeof(float),
               hipMemcpyHostToDevice);
  } else {
  // if use fp16
    printf("@@@ convert fp32 scale/bias to fp16 GroupNormPlugin\n");
    std::vector<half> scale_half(scale_.size());
    std::vector<half> bias_half(bias_.size());
    printf("@@@ scale_half.size:%d\n",scale_half.size());
    for(int i=0;i<scale_.size();++i){
      scale_half[i]=static_cast<half>(scale_[i]);
    }
    for(int i=0;i<bias_.size();++i){
      bias_half[i]=static_cast<half>(bias_[i]);
    }
    hipMalloc(&scale_gpu_, sizeof(half) * scale_half.size());
    hipMalloc(&bias_gpu_, sizeof(half) * bias_half.size());
    hipMemcpy(scale_gpu_,
               scale_half.data(),
               scale_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_half.data(),
               bias_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
  }
  return 0; 
}

bool GroupNormPlugin::supportsFormat(
    nvinfer1::DataType type, nvinfer1::PluginFormat format) const TRT_NOEXCEPT {
  if (with_fp16_) {
    return ((
             type == nvinfer1::DataType::kHALF) &&
            (format == nvinfer1::PluginFormat::kLINEAR));
  } else {
    return ((type == nvinfer1::DataType::kFLOAT) &&
            (format == nvinfer1::PluginFormat::kLINEAR));
  }
}

nvinfer1::Dims GroupNormPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *inputDims, int nbInputs) TRT_NOEXCEPT {
  return inputDims[0];
}

int GroupNormPlugin::enqueue(int batch_size,
                             const void *const *inputs,
#if IS_TRT_VERSION_LT(8000)
                             void **outputs,
                             void *workspace,
#else
                             void *const *outputs,
                             void *workspace,
#endif
                             hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = this->getInputDims(0);
  int groups = groups_;
  float eps = eps_;
  std::vector<int> input_shape;
  input_shape.push_back(batch_size);
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }
  const auto input_ddim = phi::make_ddim(input_shape);

  int C = input_shape[1];

  PADDLE_ENFORCE_EQ(
      C,
      scale_.size(),
      platform::errors::InvalidArgument(
          "scale's size should be equal to the channel number in groupnorm,"
          "but got channel number:%d, scale's size:%d.",
          C,
          scale_.size()));
  PADDLE_ENFORCE_EQ(
      C,
      bias_.size(),
      platform::errors::InvalidArgument(
          "bias's size should be equal to the channel number in groupnorm,"
          "but got channel number:%d, bias's size:%d.",
          C,
          bias_.size()));
  float *mean_d = static_cast<float *>(workspace);
  float *variance_d = mean_d+input_shape[0]*groups_;
  float *temp_variance_d = variance_d+input_shape[0]*groups_;
  hipMemset(mean_d,0,input_shape[0]*groups_*sizeof(float));
  hipMemset(variance_d,0,input_shape[0]*groups_*sizeof(float));
  hipMemset(temp_variance_d,0,input_shape[0]*groups_*sizeof(float));


  // phi::DenseTensor mean_t;
  // phi::DenseTensor variance_t;
  // phi::DenseTensor temp_variance_t;
  auto input_type = getDataType();
  if (input_type == nvinfer1::DataType::kFLOAT){
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp32";
    const float *input = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<float> group_norm;
    group_norm(stream,
             input,
             input_shape,
             reinterpret_cast<float *>(bias_gpu_),
             reinterpret_cast<float *>(scale_gpu_),
             temp_variance_d,
             groups_,
             eps_,
             output,
             mean_d,
             variance_d,
             DataLayout::kNCHW);
  } else if (input_type == nvinfer1::DataType::kHALF){
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp16";
    const half *input = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<half, float> group_norm;
    group_norm(stream,
             input,
             input_shape,
             reinterpret_cast<const half*>(bias_gpu_),
             reinterpret_cast<const half*>(scale_gpu_),
             temp_variance_d,
             groups_,
             eps_,
             output,
             mean_d,
             variance_d,
             DataLayout::kNCHW);
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The GroupNorm TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
nvinfer1::DimsExprs GroupNormPluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputDims,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  return inputDims[0];
}

bool GroupNormPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of groupnorm plugin shoule not be nullptr."));
  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));
  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
      return ((in.type == nvinfer1::DataType::kHALF) &&
              (in.format == nvinfer1::PluginFormat::kLINEAR));
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType GroupNormPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(index,
                    0,
                    platform::errors::InvalidArgument(
                        "The groupnorm Plugin only has one input, so the "
                        "index value should be 0, but get %d.",
                        index));
    PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT ||
                     input_types[0] == nvinfer1::DataType::kHALF),
                     true,
                     platform::errors::InvalidArgument(
                        "The input type should be half or float"));

  return input_types[0];
}
int GroupNormPluginDynamic::initialize() TRT_NOEXCEPT {
printf("@@@ scale cpu \n");
for(int i=0;i<5;i++){
  printf("%f, ",scale_[i]);
}
printf("\n");
if(with_fp16_==false){
  // if use fp32
    hipMalloc(&scale_gpu_, sizeof(float)*scale_.size());
    hipMalloc(&bias_gpu_, sizeof(float)*bias_.size());
    hipMemcpy(scale_gpu_,
               scale_.data(),
               scale_.size()*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_.data(),
               bias_.size()*sizeof(float),
               hipMemcpyHostToDevice);
  } else {
  // if use fp16
    printf("@@@ convert fp32 scale/bias to fp16 GroupNormDynamicPlugin\n");
    std::vector<half> scale_half(scale_.size());
    std::vector<half> bias_half(bias_.size());
    for(int i=0;i<scale_.size();++i){
      scale_half[i]=static_cast<half>(scale_[i]);
    }
    for(int i=0;i<bias_.size();++i){
      bias_half[i]=static_cast<half>(bias_[i]);
    }
    hipMalloc(&scale_gpu_, sizeof(half)*scale_.size());
    hipMalloc(&bias_gpu_, sizeof(half)*bias_.size());
    hipMemcpy(scale_gpu_,
               scale_half.data(),
               scale_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
    hipMemcpy(bias_gpu_,
               bias_half.data(),
               bias_half.size()*sizeof(half),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("@@@ scale half gpu in init \n");
    print_float<half><<<1,1>>>(reinterpret_cast<half*>(scale_gpu_),0,5,49,1);
    hipDeviceSynchronize();
    printf("\n");
  }
  return 0;
}

int GroupNormPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {
    if(with_fp16_){
      hipDeviceSynchronize();
      printf("@@@ input half gpu in enqueue begin \n");
      print_float<half><<<1,1>>>(reinterpret_cast<const half*>(inputs[0]),0,5,49,1);
      hipDeviceSynchronize();
      printf("\n");
    }
  const auto &input_dims = input_desc[0].dims;
  int groups = groups_;
  float eps = eps_;

  std::vector<int> input_shape;
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }

  const auto input_ddim = phi::make_ddim(input_shape);

  int C = input_shape[1];
  int image_size = input_shape[2] * input_shape[3];
  int batchSize = input_shape[0];
  std::vector<int64_t> batched_mean_shape = {batchSize * mean_shape_[0]};
  std::vector<int64_t> batched_variance_shape = {batchSize *
                                                 variance_shape_[0]};
  PADDLE_ENFORCE_EQ(
      C,
      scale_.size(),
      platform::errors::InvalidArgument(
          "scale's size should be equal to the channel number in groupnorm,"
          "but got feature_size:%d, scale's size:%d.",
          C,
          scale_.size()));
  PADDLE_ENFORCE_EQ(
      C,
      bias_.size(),
      platform::errors::InvalidArgument(
          "bias's size should be equal to the channel number in groupnorm,"
          "but got feature_size:%d, bias's size:%d.",
          C,
          bias_.size()));

  // phi::DenseTensor mean_t;
  // phi::DenseTensor variance_t;
  // phi::DenseTensor temp_variance_t;
  float *mean_d = static_cast<float *>(workspace);
  float *variance_d = mean_d+input_shape[0]*groups_;
  float *temp_variance_d = variance_d+input_shape[0]*groups_;
  hipMemset(mean_d,0,input_shape[0]*groups_*sizeof(float));
  hipMemset(variance_d,0,input_shape[0]*groups_*sizeof(float));
  hipMemset(temp_variance_d,0,input_shape[0]*groups_*sizeof(float));
  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp32";
    const float *input = reinterpret_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    phi::GroupNormDirectCUDAFunctor<float,float> group_norm;
    group_norm(stream,
               input,
               input_shape,
               reinterpret_cast<float*>(bias_gpu_),
               reinterpret_cast<float*>(scale_gpu_),
               temp_variance_d,
               groups,
               eps,
               output,
               mean_d,
               variance_d,
               DataLayout::kNCHW);
  } else if (input_type == nvinfer1::DataType::kHALF) {
    VLOG(1) << "TRT Plugin DataType selected. GroupNorm-->fp16";
    const half *input = reinterpret_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);

    hipDeviceSynchronize();
    printf("@@@ input half gpu in enqueue \n");
    print_float<half><<<1,1>>>(reinterpret_cast<const half*>(input),0,5,49,1);
    hipDeviceSynchronize();
    printf("\n");

    phi::GroupNormDirectCUDAFunctor<half,float> group_norm;
    group_norm(stream,
               input,
               input_shape,
               reinterpret_cast<half*>(bias_gpu_),
               reinterpret_cast<half*>(scale_gpu_),
               temp_variance_d,
               groups,
               eps,
               output,
               mean_d,
               variance_d,
               DataLayout::kNCHW);
  } else {
    // input not float
    PADDLE_THROW(platform::errors::Fatal(
        "The Groupnorm TRT Plugin's only support fp32 input"));
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
