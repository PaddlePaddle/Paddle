#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/inference/tensorrt/plugin/group_norm_op_plugin.h"

#include "paddle/phi/kernels/group_norm_kernel.h"
#include "paddle/phi/kernels/gpu/group_norm_utils.h"

#include "paddle/phi/common/layout.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
    using DataLayout = framework::DataLayout;
    template <typename T>
    __global__ void GroupNormForward(const T* x,
                                    const T* mean,
                                    const T* var,
                                    const T* scale,
                                    const T* bias,
                                    int N,
                                    int C,
                                    int W,
                                    int imsize,
                                    int groups,
                                    int group_size,
                                    T epsilon,
                                    T* y,
                                    T* real_var) {
        int gid = blockIdx.y;
        int cid = blockIdx.x;
        int bid = blockIdx.z;
        int H = imsize / W;
        int ccid = gid * group_size + cid;
        if (ccid >= C) return;
        auto ng = bid * groups + gid;
        T x_mean = mean[ng];
        T x_var = var[ng];
        x_var = x_var - x_mean * x_mean;
        T var_inv = rsqrt(x_var + epsilon);
        if (cid == 0 && threadIdx.x == 0) {
            real_var[ng] = x_var;
        }
        for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
            T val;
            int hid, wid;
            int index = (bid * C + ccid) * imsize + imid;
            val = x[index];
            /*
            if (data_layout == DataLayout::kNCHW) {
                val = x[index];
            } else {
                hid = imid / W;
                wid = imid % W;
                val = x[(bid * H + hid) * W * C + wid * C + ccid];
            }
            */
            val = (val - x_mean) * var_inv;

            val *= scale[ccid];
            
            val += bias[ccid];
            y[index] = val;
            /*
            if (data_layout == DataLayout::kNCHW) {
                y[index] = val;
            } else {
                y[(bid * H + hid) * W * C + wid * C + ccid] = val;
            }
            */
        }
    }

nvinfer1::DimsExprs GroupNormPluginDynamic::getOutputDimensions(
        int output_index,
        const nvinfer1::DimsExprs *inputDims,
        int nb_inputs,
        nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  return inputDims[0];
}

bool GroupNormPluginDynamic::supportsFormatCombination(
        int pos,
        const nvinfer1::PluginTensorDesc *in_out,
        int nb_inputs,
        int nb_outputs) TRT_NOEXCEPT {
    PADDLE_ENFORCE_NOT_NULL(
        in_out,
        platform::errors::InvalidArgument(
            "The input of groupnorm plugin shoule not be nullptr."));
    PADDLE_ENFORCE_LT(
        pos,
        nb_inputs + nb_outputs,
        platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                            "num(%d) of the input and the output.",
                                            pos,
                                            nb_inputs + nb_outputs));
    const nvinfer1::PluginTensorDesc &in = in_out[pos];
    if (pos == 0) {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
    const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
    // output
    return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType GroupNormPluginDynamic::getOutputDataType(
        int index,
        const nvinfer1::DataType * input_types,
        int nb_inputs) const TRT_NOEXCEPT{
    PADDLE_ENFORCE_EQ(index,
        0,
        platform::errors::InvalidArgument(
            "The groupnorm Plugin only has one input, so the "
            "index value should be 0, but get %d.",
            index));
    return input_types[0];

}

int GroupNormPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    const void* const* inputs,
    //TODO wangbojun check for trt > version 8000
    //TODO void ** outputs should work for trt < 8000
    void * const *outputs,
    void* workspace,
    hipStream_t stream) TRT_NOEXCEPT {
        const auto &input_dims=input_desc[0].dims;
        int groups=groups_;
        float eps=eps_;

        std::vector<int> input_shape;
        for (int i=0;i<input_dims.nbDims;i++){
            input_shape.push_back(input_dims.d[i]);
        }

        const auto input_ddim = phi::make_ddim(input_shape);
        auto matrix_dim = phi::flatten_to_2d(input_ddim, 1); //check
        int feature_size = static_cast<int>(matrix_dim[1]);
        PADDLE_ENFORCE_EQ(feature_size,
                          scale_.size(),
                          platform::errors::InvalidArgument(
                              "scale's size should be equal to the feature_size in groupnorm,"
                              "but got feature_size:%d, scale's size:%d.",
                              feature_size,
                              scale_.size()));
        PADDLE_ENFORCE_EQ(feature_size,
                          bias_.size(),
                          platform::errors::InvalidArgument(
                              "bias's size should be equal to the feature_size in groupnorm,"
                              "but got feature_size:%d, bias's size:%d.",
                              feature_size,
                              bias_.size()));


        int device_id;
        hipGetDevice(&device_id);
      
        auto input_type = input_desc[0].type;
        if (input_type == nvinfer1::DataType::kFLOAT) {
            const float *input = reinterpret_cast<const float *>(inputs[0]);
            float *output = static_cast<float *>(outputs[0]);
        
            scale_t.Resize(phi::make_ddim({feature_size}));
            bias_t.Resize(phi::make_ddim({feature_size}));
            
            mean_t.Resize(phi::make_ddim(mean_shape_));
            variance_t.Resize(phi::make_ddim(variance_shape_));
            framework::Tensor temp_variance_t;
            temp_variance_t.Resize(phi::make_ddim(variance_shape_));
            float *scale_d =
                scale_t.mutable_data<float>(platform::CUDAPlace(device_id));
            float *bias_d = bias_t.mutable_data<float>(platform::CUDAPlace(device_id));
            float *mean_d = mean_t.mutable_data<float>(platform::CUDAPlace(device_id));
            float *variance_d =
                variance_t.mutable_data<float>(platform::CUDAPlace(device_id));
            float * temp_variance_d=temp_variance_t.mutable_data<float>(platform::CUDAPlace(device_id));
            hipMemcpyAsync(scale_d,
                            scale_.data(),
                            sizeof(float) * feature_size,
                            hipMemcpyHostToDevice,
                            stream);
            hipMemcpyAsync(bias_d,
                            bias_.data(),
                            sizeof(float) * feature_size,
                            hipMemcpyHostToDevice,
                            stream);
            const auto input_ddim=phi::make_ddim(input_shape);
            const int C = input_ddim[1];
            const int group_size = C/groups_;
            const int W=input_ddim[input_ddim.size()-1];
            int image_size=1;
            for (int i=2;i<input_ddim.size();++i){
                image_size*=input_ddim[i];
            }
            int block_size=std::min(1024,image_size);
            dim3 grid(group_size,groups_,input_ddim[0]);
            dim3 threads(block_size,1,1);
            using AccT = typename phi::kps::details::MPTypeTrait<float>::Type;
            constexpr int vec_size=sizeof(float4)/sizeof(float);
            int size=group_size*image_size; // group element size
            const int max_num_threads=1024;
            int max_block_size = std::min(size/vec_size,max_num_threads);
            int block_size_nchw=1;
            while(block_size_nchw<max_block_size){
                block_size_nchw*=2;
            }

            block_size_nchw=std::max(block_size_nchw,phi::kps::details::kWarpSize);
            dim3 grids(input_ddim[0]*groups_);
            dim3 blocks(block_size_nchw);
            if (size<vec_size*block_size_nchw){
                phi::ScalarGetMeanAndVarNCHW<float><<<grids, blocks, 0,stream>>>(
                    input, mean_d, temp_variance_d, size);
            } else {
                phi::VectorizedGetMeanAndVarNCHW<float, AccT, vec_size>
                <<<grids,blocks,0,stream>>>(
                    input,mean_d,temp_variance_d,size);
            }
            //int flags =
            //    (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
            GroupNormForward<float><<<grids,threads,0,stream>>>(
                input,
                mean_d,
                temp_variance_d,
                scale_d,
                bias_d,
                input_ddim[0],
                C,
                W,
                image_size,
                groups_,
                group_size,
                eps_,
                output,
                variance_d);
            /*
            UNROLL_ALL_CASES(3,
              GroupNormForward,
              input,
              mean_d,
              temp_variance_d,
              scale_d,
              bias_d,
              x_dims[0],
              C,
              W,
              imsize,
              groups,
              group_size,
              eps_,
              y,
              variance_d,
              DataLayout::kNCHW); // only support NCHW
              */
        } else {
            // input not float
            PADDLE_THROW(platform::errors::Fatal(
                "The Groupnorm TRT Plugin's only support fp16 input"));        
        }
        return hipGetLastError() != hipSuccess;
    }



} // plugin
} // tenssort
} // inference
} // paddle