// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/skip_layernorm_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

int SkipLayerNormPluginDynamic::initialize() {
  hipMalloc(&bias_gpu_, sizeof(float) * bias_size_);
  hipMemcpy(bias_gpu_, bias_.data(), bias_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&scale_gpu_, sizeof(float) * scale_size_);
  hipMemcpy(scale_gpu_, scale_.data(), scale_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  return 0;
}

nvinfer1::DimsExprs SkipLayerNormPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  PADDLE_ENFORCE_EQ(
      inputs[0].nbDims, 5,
      platform::errors::InvalidArgument(
          "The Input dim of the SkipLayernorm should be 5, but it's (%d) now.",
          inputs[0].nbDims));
  return inputs[0];
}

bool SkipLayerNormPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    if (ban_fp16_) {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT ||
              in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType SkipLayerNormPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0,
                    platform::errors::InvalidArgument(
                        "The SkipLayerNorm Plugin only has one input, so the "
                        "index value should be 0, but get %d.",
                        index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT ||
                     input_types[0] == nvinfer1::DataType::kHALF),
                    true, platform::errors::InvalidArgument(
                              "The input type should be half or float"));
  return input_types[0];
}

int SkipLayerNormPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  size_t num = ProductDim(input_dims);
  int hidden = input_dims.d[2];

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    const float *input1 = static_cast<const float *>(inputs[0]);
    const float *input2 = static_cast<const float *>(inputs[1]);
    float *output = static_cast<float *>(outputs[0]);
    operators::math::SkipLayerNormFunctor<float> skip_layer_norm_func;
    skip_layer_norm_func(num, hidden, input1, input2, scale_gpu_, bias_gpu_,
                         output, eps_, stream);
  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    const half *input1 = static_cast<const half *>(inputs[0]);
    const half *input2 = static_cast<const half *>(inputs[1]);
    half *output = static_cast<half *>(outputs[0]);
    operators::math::SkipLayerNormFunctor<half> skip_layer_norm_func;
    skip_layer_norm_func(num, hidden, input1, input2, scale_gpu_, bias_gpu_,
                         output, static_cast<half>(eps_), stream);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The SkipLayerNorm TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
