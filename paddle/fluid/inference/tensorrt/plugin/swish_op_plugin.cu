#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/swish_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

SwishPlugin *CreateSwishPluginDeserialize(const void *buffer, size_t length) {
  return new SwishPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("swish_plugin", CreateSwishPluginDeserialize);

int SwishPlugin::initialize() { return 0; }

nvinfer1::Dims SwishPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

template <typename T>
__device__ T math_exp(T a);

#ifdef SUPPORTS_CUDA_FP16
template <>
__device__ half math_exp<half>(half a) {
  return hexp(a);
}
#endif

template <>
__device__ float math_exp<float>(float a) {
  return expf(a);
}

template <typename T>
__global__ void swish_kernel(int num, const T *input, T *output, T beta) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
#if __CUDA_ARCH__ >= 350
    output[index] =
        __ldg(input + index) /
        (static_cast<T>(1.0) + math_exp<T>(-beta * __ldg(input + index)));
#else
    output[index] = input[index] /
                    (static_cast<T>(1.0) + math_exp<T>(-beta * input[index]));
#endif
  }
}

int SwishPlugin::enqueue(int batch_size, const void *const *inputs,
                         void **outputs, void *workspace, hipStream_t stream) {
  // input dims is CHW.
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  float *output = reinterpret_cast<float **>(outputs)[0];
  int num = batch_size;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;
  swish_kernel<<<blocks, threads, 0, stream>>>(num, input, output, beta_);

  return hipGetLastError() != hipSuccess;
}

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

int SwishPluginDynamic::initialize() {
  setPluginNamespace("swish");
  getPluginNamespace();
  return 0;
}

size_t SwishPluginDynamic::getSerializationSize() const { return 0; }

void SwishPluginDynamic::serialize(void *buffer) const {}

nvinfer1::DimsExprs SwishPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  return inputs[0];
}

bool SwishPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    return (in.type == nvinfer1::DataType::kFLOAT ||
            in.type == nvinfer1::DataType::kHALF) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType SwishPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The Swish Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  return input_types[0];
}

int SwishPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                                const nvinfer1::PluginTensorDesc *output_desc,
                                const void *const *inputs, void *const *outputs,
                                void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  size_t num = ProductDim(input_dims);
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    const float *input = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    swish_kernel<float><<<blocks, threads, 0, stream>>>(num, input, output,
                                                        beta_);
  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    const half *input = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    swish_kernel<half><<<blocks, threads, 0, stream>>>(
        num, input, output, static_cast<half>(beta_));
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The Swish TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
