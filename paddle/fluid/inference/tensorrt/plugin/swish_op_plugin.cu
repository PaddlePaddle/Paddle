#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/swish_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

SwishPlugin *CreateSwishPluginDeserialize(const void *buffer, size_t length) {
  return new SwishPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("swish_plugin", CreateSwishPluginDeserialize);

int SwishPlugin::initialize() { return 0; }

nvinfer1::Dims SwishPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}
__global__ void swish_kernel(int num, const float *input, float *output,
                             float beta) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < num) {
#if __CUDA_ARCH__ >= 350
    output[index] =
        __ldg(input + index) / (1.0f + expf(-beta * __ldg(input + index)));
#else
    output[index] = input[index] / (1.0f + expf(-beta * input[index]));
#endif
  }
}

int SwishPlugin::enqueue(int batch_size, const void *const *inputs,
                         void **outputs, void *workspace, hipStream_t stream) {
  // input dims is CHW.
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  float *output = reinterpret_cast<float **>(outputs)[0];
  int num = batch_size;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  int threads = 1024;
  int blocks = (num + threads - 1) / threads;
  swish_kernel<<<blocks, threads, 0, stream>>>(num, input, output, beta_);

  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
