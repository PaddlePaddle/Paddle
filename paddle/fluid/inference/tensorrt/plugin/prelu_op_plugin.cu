// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/prelu_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/operators/math/prelu.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

PReluPlugin *CreatePreluPluginDeserialize(const void *buffer, size_t length) {
  return new PReluPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("prelu_plugin", CreatePreluPluginDeserialize);

int PReluPlugin::initialize() {
  hipMalloc(&p_gpu_weight_, sizeof(float) * weight_.size());
  hipMemcpy(p_gpu_weight_, weight_.data(), weight_.size() * sizeof(float),
             hipMemcpyHostToDevice);
}

nvinfer1::Dims PReluPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

int PReluPlugin::enqueue(int batch_size, const void *const *inputs,
                         void **outputs, void *workspace, hipStream_t stream) {
  // input dims is CHW.
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  // const float *alpha = reinterpret_cast<const float *>(alpha_.get().values);
  const float *alpha = p_gpu_weight_;
  float *output = reinterpret_cast<float **>(outputs)[0];

  std::vector<int> input_shape;
  input_shape.push_back(batch_size);
  for (int i = 0; i < input_dims.nbDims; i++) {
    input_shape.push_back(input_dims.d[i]);
  }

  if (mode_ == "channel") {
    operators::math::PreluChannelWiseDirectCUDAFunctor<float>
        prelu_channel_wise;
    prelu_channel_wise(stream, input, alpha, output, input_shape);
  } else if (mode_ == "element") {
    operators::math::PreluElementWiseDirectCUDAFunctor<float>
        prelu_element_wise;
    prelu_element_wise(stream, input, alpha, output, input_shape);
  } else {
    operators::math::PreluScalarDirectCUDAFunctor<float> prelu_scalar;
    prelu_scalar(stream, input, alpha, output, input_shape);
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
