// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>

#include <cassert>
#include <vector>

#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/prelu_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/operators/math/prelu.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

PReluPlugin *CreatePreluPluginDeserialize(const void *buffer, size_t length) {
  return new PReluPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("prelu_plugin", CreatePreluPluginDeserialize);

int PReluPlugin::initialize() {
  hipMalloc(&p_gpu_weight_, sizeof(float) * weight_.size());
  hipMemcpy(p_gpu_weight_, weight_.data(), weight_.size() * sizeof(float),
             hipMemcpyHostToDevice);
  return 0;
}

void PReluPlugin::terminate() {
  if (p_gpu_weight_) {
    hipFree(p_gpu_weight_);
    p_gpu_weight_ = nullptr;
  }
}

nvinfer1::Dims PReluPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

int PReluPlugin::enqueue(int batch_size, const void *const *inputs,
                         void **outputs, void *workspace, hipStream_t stream) {
  // input dims is CHW.
  const auto &input_dims = this->getInputDims(0);
  const float *input = reinterpret_cast<const float *>(inputs[0]);
  // const float *alpha = reinterpret_cast<const float *>(alpha_.get().values);
  const float *alpha = p_gpu_weight_;
  float *output = reinterpret_cast<float **>(outputs)[0];
  int numel = 1;
  for (int i = 0; i < input_dims.nbDims; i++) {
    numel *= input_dims.d[i];
  }

  if (mode_ == "channel") {
    operators::math::PreluChannelWiseDirectCUDAFunctor<float>
        prelu_channel_wise;
    prelu_channel_wise(stream, input, alpha, output, input_dims.d[0],
                       input_dims.d[1], numel);
  } else if (mode_ == "element") {
    operators::math::PreluElementWiseDirectCUDAFunctor<float>
        prelu_element_wise;
    prelu_element_wise(stream, input, alpha, output, input_dims.d[0], numel);
  } else {
    operators::math::PreluScalarDirectCUDAFunctor<float> prelu_scalar;
    prelu_scalar(stream, input, alpha, output, numel);
  }
  return hipGetLastError() != hipSuccess;
}

#if IS_TRT_VERSION_GE(6000)

void PReluPluginDynamic::terminate() {
  if (p_gpu_weight_) {
    hipFree(p_gpu_weight_);
  }
}

int PReluPluginDynamic::initialize() {
  hipMalloc(&p_gpu_weight_, sizeof(float) * weight_.size());
  hipMemcpy(p_gpu_weight_, weight_.data(), weight_.size() * sizeof(float),
             hipMemcpyHostToDevice);
  return 0;
}
size_t PReluPluginDynamic::getSerializationSize() const { return 0; }

void PReluPluginDynamic::serialize(void *buffer) const {}

nvinfer1::DimsExprs PReluPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  return inputs[0];
}

bool PReluPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  return ((in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
          in_out[pos].format == nvinfer1::PluginFormat::kNCHW);
}

nvinfer1::DataType PReluPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The PRelu Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT), true,
                    platform::errors::InvalidArgument(
                        "The input type should be half or float"));
  return input_types[0];
}

int PReluPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                                const nvinfer1::PluginTensorDesc *output_desc,
                                const void *const *inputs, void *const *outputs,
                                void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  const float *alpha = p_gpu_weight_;
  const float *input = static_cast<const float *>(inputs[0]);
  float *output = static_cast<float *>(outputs[0]);
  int numel = 1;
  for (int i = 0; i < input_dims.nbDims; i++) {
    numel *= input_dims.d[i];
  }

  if (mode_ == "channel") {
    operators::math::PreluChannelWiseDirectCUDAFunctor<float>
        prelu_channel_wise;
    prelu_channel_wise(stream, input, alpha, output, input_dims.d[0],
                       input_dims.d[1], numel);
  } else if (mode_ == "element") {
    operators::math::PreluElementWiseDirectCUDAFunctor<float>
        prelu_element_wise;
    prelu_element_wise(stream, input, alpha, output, input_dims.d[0], numel);
  } else {
    operators::math::PreluScalarDirectCUDAFunctor<float> prelu_scalar;
    prelu_scalar(stream, input, alpha, output, numel);
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
