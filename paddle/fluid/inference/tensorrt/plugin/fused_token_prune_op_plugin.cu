#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/inference/tensorrt/plugin/fused_token_prune_op_plugin.h"
#include "paddle/fluid/operators/fused_token_prune_op.cu.h"
#include "paddle/fluid/platform/device_context.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

#if IS_TRT_VERSION_GE(6000)

template <typename T>
__global__ void ElementwiseMask(const T* a, const T* b, T* res, int num_raws,
                                int num_cols) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * num_cols) return;
  const T zero = 0;
  res[tid] = b[tid] >= zero ? a[tid] : zero;
}

__global__ void FillIndex(int* indices, int num_rows, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_rows * num_cols) return;

  int col = tid % num_cols;
  int row = tid / num_cols;

  indices[tid] = col;
}

template <typename T>
__global__ void ReduceSum(const T* src, T* dst, int bsz, int nb_head,
                          int max_seq_len) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= bsz * max_seq_len) return;
  int batch = tid / max_seq_len, col = tid % max_seq_len;
  auto* src_start = src + batch * (nb_head * max_seq_len * max_seq_len) + col;
  auto* dst_addr = dst + batch * max_seq_len + col;
  *dst_addr = 0;
  for (int i = 0; i < nb_head * max_seq_len; ++i) {
    *dst_addr += *(src_start + i * max_seq_len);
  }
}

template <typename T>
__global__ void SlicedArgsort(T* data, int* indices, int num_rows,
                              int num_cols) {
  auto raw = blockIdx.x * blockDim.x + threadIdx.x;
  if (raw >= num_rows) return;
  thrust::sort_by_key(thrust::seq, data + raw * num_cols + 1,
                      data + (raw + 1) * num_cols, indices + raw * num_cols + 1,
                      thrust::greater<T>());
}

template <typename T>
__global__ void TakeAlongLastAxis2D(const T* src, T* dst, int* indices,
                                    int num_rows, int src_num_cols,
                                    int dst_num_cols, int num_elements) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_rows * dst_num_cols) return;

  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  for (int i = 0; i < num_elements; ++i) {
    dst[tid * num_elements + i] =
        *(src +
          (raw * src_num_cols + indices[raw * src_num_cols + col]) *
              num_elements +
          i);
  }
}

nvinfer1::DimsExprs FusedTokenPrunePluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs* inputs, int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(
      output_index, 0,
      platform::errors::InvalidArgument(
          "The FusedTokenPrune Plugin only has one output, so the "
          "index value should be 0, but get %d.",
          output_index));
  auto attn_dims = inputs[0], x_dims = inputs[1];
  int max_seq_len = attn_dims.d[2]->getConstantValue();
  int slimmed_seq_len = max_seq_len * factor_;
  nvinfer1::DimsExprs ret = x_dims;
  ret.d[1] = expr_builder.constant(slimmed_seq_len);
  return ret;
}

bool FusedTokenPrunePluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* in_out, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (pos < 3) {
    if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
      return (in.type == nvinfer1::DataType::kFLOAT ||
              in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc& prev = in_out[pos - 1];

  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType FusedTokenPrunePluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(
      index, 0,
      platform::errors::InvalidArgument(
          "The EmbEltwiseLayernorm Plugin only has one output, so the "
          "index value should be 0, but get %d.",
          index));
  return input_types[1];
}

template <typename T>
int FusedTokenPrunePluginDynamic::enqueueImpl(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc, const void* const* inputs,
    void* const* outputs, void* workspace, hipStream_t stream,
    T* attn_tmp_data, T* attn_by_data, int device_id) {
  auto attn_dims = input_desc[0].dims;
  auto x_dims = input_desc[1].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  auto c = x_dims.d[2];

  const T* attn_data = static_cast<const T*>(inputs[0]);
  const T* x_data = static_cast<const T*>(inputs[1]);
  const T* mask_data = static_cast<const T*>(inputs[2]);
  T* output_data = static_cast<T*>(outputs[0]);

  int total = bsz * nb_head * max_seq_len * max_seq_len;
  int block = operators::ComputeBlockSize(max_seq_len);
  int grid = operators::CeilDivide(total, block);
  ElementwiseMask<T><<<grid, block, 0, stream>>>(
      attn_data, mask_data, attn_tmp_data, grid, max_seq_len);

  auto* device_ctx = static_cast<platform::CUDADeviceContext*>(
      platform::DeviceContextPool::Instance().Get(
          platform::CUDAPlace(device_id)));
  const platform::CUDADeviceContext& dev_ctx = *device_ctx;
  // const std::vector<int64_t> reduce_dims{1, 2};
  // phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
  //     dev_ctx, attn_tmp, false, reduce_dims, false, attn_by.dtype(),
  //     &attn_by);
  total = bsz * max_seq_len;
  grid = operators::CeilDivide(total, block);
  ReduceSum<T><<<grid, block, 0, stream>>>(attn_tmp_data, attn_by_data, bsz,
                                           nb_head, max_seq_len);

  framework::Tensor attn_by_indices;
  attn_by_indices.Resize({bsz, max_seq_len});
  auto* attn_by_indices_data =
      attn_by_indices.mutable_data<int>(platform::CUDAPlace(device_id));

  FillIndex<<<grid, block, 0, stream>>>(attn_by_indices_data, bsz, max_seq_len);
  SlicedArgsort<T><<<bsz, 1, 0, stream>>>(attn_by_data, attn_by_indices_data,
                                          bsz, max_seq_len);

  int slimmed_x_len = max_seq_len * factor_;
  // auto slimmed_indices = phi::funcs::Slice<int>(dev_ctx, attn_by_indices,
  // {1},
  //                                                 {0}, {slimmed_x_len});
  block = operators::ComputeBlockSize(slimmed_x_len);
  TakeAlongLastAxis2D<T><<<grid, block, 0, stream>>>(
      x_data, output_data, attn_by_indices_data, bsz, max_seq_len,
      slimmed_x_len, c);
}

int FusedTokenPrunePluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc, const void* const* inputs,
    void* const* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT {
  auto input_type = input_desc[0].type;
  auto attn_dims = input_desc[0].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  int device_id;
  hipGetDevice(&device_id);
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp32";

    framework::Tensor attn_tmp;
    attn_tmp.Resize({bsz, nb_head, max_seq_len, max_seq_len});
    auto* attn_tmp_data =
        attn_tmp.mutable_data<float>(platform::CUDAPlace(device_id));

    framework::Tensor attn_by;
    attn_by.Resize({bsz, max_seq_len});
    auto* attn_by_data =
        attn_by.mutable_data<float>(platform::CUDAPlace(device_id));

    enqueueImpl<float>(input_desc, output_desc, inputs, outputs, workspace,
                       stream, attn_tmp_data, attn_by_data, device_id);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp16";

    framework::Tensor attn_tmp;
    attn_tmp.Resize({bsz, nb_head, max_seq_len, max_seq_len});
    auto* attn_tmp_data_tmp = attn_tmp.mutable_data<int16_t>(
        platform::CUDAPlace(device_id));  // NOLINT
    auto* attn_tmp_data = reinterpret_cast<half*>(attn_tmp_data_tmp);

    framework::Tensor attn_by;
    attn_by.Resize({bsz, max_seq_len});
    auto* attn_by_data_tmp =
        attn_by.mutable_data<int16_t>(platform::CUDAPlace(device_id));
    auto* attn_by_data = reinterpret_cast<half*>(attn_by_data_tmp);

    enqueueImpl<half>(input_desc, output_desc, inputs, outputs, workspace,
                      stream, attn_tmp_data, attn_by_data, device_id);

#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) TensorRT Plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The FusedTokenPrune TRT Plugin's input type "
                                "should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

#endif
}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
