#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>

#include "hipcub/hipcub.hpp"

#include "paddle/phi/kernels/funcs/math_function.h"

#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
<<<<<<< HEAD
#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
=======
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/device_context.h"
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81

#include "paddle/fluid/inference/tensorrt/plugin/fused_token_prune_op_plugin.h"
#include "paddle/fluid/operators/fused_token_prune_op.cu.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

<<<<<<< HEAD
__global__ void compute_token_length(const int32_t* src,
                                     int32_t* dst,
                                     float scale) {
  int32_t it = threadIdx.x;
  dst[it] = max(static_cast<int>((src[it + 1] - src[it]) * scale), 1);
}

template <typename T>
__global__ void fill_index_padding_score(int32_t* token_index,
                                         const T* scores,
                                         int32_t sequnce_length,
                                         T* padding_scores) {
  int padding_scores_it = threadIdx.x + blockIdx.x * blockDim.x;
  int scores_it = threadIdx.x + blockIdx.x * sequnce_length;
  token_index[padding_scores_it] = threadIdx.x;
  if (threadIdx.x < sequnce_length) {
    padding_scores[padding_scores_it] = scores[scores_it];
  } else {
    padding_scores[padding_scores_it] = 0;
  }
}

template <typename T, int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void general_topk_pair_sort(T* in_keys, int32_t* in_out_values) {
  typedef hipcub::BlockRadixSort<T, BLOCK_THREADS, ITEMS_PER_THREAD, int>
      BlockRadixSort;
  typedef cub::
      BlockLoad<T, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE>
          BlockLoadKey;
  typedef cub::
      BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE>
          BlockLoadValue;
  typedef cub::
      BlockStore<T, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE>
          BlockStoreKey;
  typedef hipcub::BlockStore<int,
                          BLOCK_THREADS,
                          ITEMS_PER_THREAD,
                          hipcub::BLOCK_STORE_TRANSPOSE>
      BlockStoreValue;

  __shared__ union {
    typename BlockRadixSort::TempStorage sort;
    typename BlockLoadKey::TempStorage loadkey;
    typename BlockLoadValue::TempStorage loadvalue;
    typename BlockStoreKey::TempStorage storekey;
    typename BlockStoreValue::TempStorage storevalue;
  } temp_storage;

  int block_offset = blockIdx.x * BLOCK_THREADS * ITEMS_PER_THREAD;

  T thread_keys[ITEMS_PER_THREAD];
  int thread_values[ITEMS_PER_THREAD];
  BlockLoadKey(temp_storage.loadkey).Load(in_keys + block_offset, thread_keys);
  BlockLoadValue(temp_storage.loadvalue)
      .Load(in_out_values + block_offset, thread_values);
  __syncthreads();

  BlockRadixSort(temp_storage.sort).SortDescending(thread_keys, thread_values);
  __syncthreads();

  BlockStoreValue(temp_storage.storevalue)
      .Store(in_out_values + block_offset, thread_values);
}

__global__ void varlen_prune_token_change_order(
    const half* tokens,
    const int32_t* token_pos,
    const int32_t padding_token_length,
    const int32_t* token_index,
    half* output) {
  int batch = blockIdx.x;
  int token_it = batch * gridDim.y + blockIdx.y;
  int pre_value_it =
      token_it * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
  int token_index_it = batch * padding_token_length + blockIdx.y;

  if (token_index[token_index_it] < token_pos[batch + 1] - token_pos[batch]) {
    output[(token_index[token_index_it] + token_pos[batch]) * gridDim.z *
               blockDim.x +
           blockIdx.z * blockDim.x + threadIdx.x] = tokens[pre_value_it];
  }
}

template <typename T>
__global__ void prune_token_change_order(const T* tokens,
                                         int32_t new_sequnce_length,
                                         const int32_t padding_token_length,
                                         const int32_t* token_index,
                                         T* output) {
  int batch = blockIdx.x;
  int token_it = batch * gridDim.y + blockIdx.y;
  int pre_value_it =
      token_it * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;
  int token_index_it = batch * padding_token_length + blockIdx.y;

  if (token_index[token_index_it] < new_sequnce_length) {
    output[(batch * new_sequnce_length + token_index[token_index_it]) *
               gridDim.z * blockDim.x +
           blockIdx.z * blockDim.x + threadIdx.x] = tokens[pre_value_it];
  }
}

template <typename T>
__global__ void prune_token_keep_order(const T* tokens,
                                       int32_t pre_sequnce_length,
                                       int32_t new_sequnce_length,
                                       const int32_t padding_token_length,
                                       const int32_t* token_index,
                                       T* output0,
                                       int32_t* output1) {
  int batch = blockIdx.x;
  int index = 0;
  for (int i = 0; i < pre_sequnce_length; ++i) {
    if (token_index[batch * padding_token_length + i] < new_sequnce_length) {
      output0[(batch * new_sequnce_length + index) * gridDim.y * blockDim.x +
              blockIdx.y * blockDim.x + threadIdx.x] =
          tokens[(batch * pre_sequnce_length + i) * gridDim.y * blockDim.x +
                 blockIdx.y * blockDim.x + threadIdx.x];
      output1[batch * new_sequnce_length + index] = i;
      index++;
    }
=======
#if IS_TRT_VERSION_GE(6000)

template <typename T>
__global__ void ElementwiseMask(const T* a,
                                const T* b,
                                T* res,
                                int num_elements) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_elements) return;
  const T zero = 0;
  res[tid] = b[tid] >= zero ? a[tid] : zero;
#endif
}

template <typename T>
__global__ void FillZero(T* data, int len) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  const T zero = 0;
  data[tid] = zero;
}

__global__ void FillIndex(int32_t* indices, int num_raws, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * num_cols) return;

  int col = tid % num_cols;
  int raw = tid / num_cols;

  indices[tid] = col;
}

template <typename T>
__global__ void MaximumFirst(T* mat, int num_raws, int num_cols, T max_value) {
  auto raw = blockIdx.x * blockDim.x + threadIdx.x;
  if (raw >= num_raws) return;
  mat[raw * num_cols] = max_value;
}

__global__ void FillOffsets(int* offsets, int num_raws, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid > num_raws) return;

  offsets[tid] = tid * num_cols;
}

template <typename T>
__global__ void Slice(
    const T* src, T* dst, int num_raws, int src_num_cols, int dst_num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * dst_num_cols) return;
  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  dst[tid] = src[raw * src_num_cols + col];
}

template <typename T>
__global__ void ReduceSum2(
    const T* src, T* dst, int bsz, int nb_head, int max_seq_len) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_blocks_per_head = ((max_seq_len / blockDim.x) * max_seq_len);
  int batch = bid / (nb_head * num_blocks_per_head);
  int col = bid % max_seq_len;
  int head = (bid / num_blocks_per_head) % nb_head;

  extern __shared__ T res_float[];
  res_float[tid] =
      src[batch * (nb_head * max_seq_len * max_seq_len) +
          head * (max_seq_len * max_seq_len) + col + tid * max_seq_len];
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      res_float[tid] += res_float[tid + offset];
    }
    __syncthreads();
    if (offset % 2 == 1 && tid == offset - 2) {
      res_float[tid] += res_float[tid + 1];
    }
  }

  if (tid == 0) {
    auto* dst_addr = dst + batch * max_seq_len + col;
    atomicAdd(dst_addr, res_float[0]);
  }
}

template <>
__global__ void ReduceSum2<half>(
    const half* src, half* dst, int bsz, int nb_head, int max_seq_len) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_blocks_per_head = ((max_seq_len / blockDim.x) * max_seq_len);
  int batch = bid / (nb_head * num_blocks_per_head);
  int col = bid % max_seq_len;
  int head = (bid / num_blocks_per_head) % nb_head;

  extern __shared__ half res_half[];
  res_half[tid] =
      src[batch * (nb_head * max_seq_len * max_seq_len) +
          head * (max_seq_len * max_seq_len) + col + tid * max_seq_len];
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      res_half[tid] += res_half[tid + offset];
    }
    __syncthreads();
    if (offset % 2 == 1 && tid == offset - 2) {
      res_half[tid] += res_half[tid + 1];
    }
    __syncthreads();
  }

  if (tid == 0) {
    platform::fastAtomicAdd<platform::float16>(
        reinterpret_cast<platform::float16*>(dst),
        static_cast<size_t>(batch * max_seq_len + col),
        static_cast<size_t>(bsz * max_seq_len),
        static_cast<platform::float16>(res_half[0]));
  }
#endif
}

template <typename T>
__global__ void TakeAlongAxis(const T* src,
                              T* dst,
                              int32_t* indices,
                              int num_raws,
                              int src_num_cols,
                              int dst_num_cols,
                              int num_elements) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * dst_num_cols) return;

  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  for (int i = 0; i < num_elements; ++i) {
    dst[tid * num_elements + i] =
        *(src + (raw * src_num_cols + indices[tid]) * num_elements + i);
  }
}

__global__ void pos_id_prune_kernel(const int32_t* src,
                                    int32_t* dst,
                                    int pos_nums,
                                    float scale) {
  dst[0] = 0;
  for (int i = 1; i < pos_nums; i++) {
    dst[i] =
        dst[i - 1] + max(static_cast<int>((src[i] - src[i - 1]) * scale), 2);
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
  }
}

nvinfer1::DimsExprs FusedTokenPrunePluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs* inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  auto x_dims = inputs[1], new_mask_dims = inputs[3];
  if (flag_varseqlen_) {
<<<<<<< HEAD
    // max sum of seqlen: ceil(sum / scale) + n -1 >= for(i=0;i<n;i++) {sum +=
    // floor(num(i) / scale)} auto
    // pruned_sum_length=std::ceil(inputs[4].d[0]*new_mask_dims.d[2]/inputs[6].d[1])+
    // inputs[1].d[0] - 1;
    auto pruned_sum_length = expr_builder.operation(
        nvinfer1::DimensionOperation::kSUB,
        *expr_builder.operation(
            nvinfer1::DimensionOperation::kSUM,
            *expr_builder.operation(
                nvinfer1::DimensionOperation::kCEIL_DIV,
                *expr_builder.operation(nvinfer1::DimensionOperation::kPROD,
                                        *inputs[4].d[0],
                                        *new_mask_dims.d[2]),
                *inputs[6].d[1]),
            *inputs[1].d[0]),
        *expr_builder.constant(1));
    if (output_index == 0) {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 4;
      ret.d[0] = pruned_sum_length;
      ret.d[1] = x_dims.d[2];
      ret.d[2] = expr_builder.constant(1);
      ret.d[3] = expr_builder.constant(1);
=======
    if (output_index == 0) {
      nvinfer1::DimsExprs ret = x_dims;
      ret.d[1] = new_mask_dims.d[2];
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
      return ret;
    } else if (output_index == 1) {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = new_mask_dims.d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else if (output_index == 2) {
      // word id
      nvinfer1::DimsExprs ret;
      ret.nbDims = 1;
<<<<<<< HEAD
      ret.d[0] = pruned_sum_length;
=======
      // max sum of seqlen: pre_seqlen * new_mask[2] / mask[1] + 2 * batchs
      const auto* two = expr_builder.constant(2);
      ret.d[0] = expr_builder.operation(
          nvinfer1::DimensionOperation::kSUM,
          *expr_builder.operation(
              nvinfer1::DimensionOperation::kFLOOR_DIV,
              *expr_builder.operation(nvinfer1::DimensionOperation::kPROD,
                                      *inputs[4].d[0],
                                      *new_mask_dims.d[2]),
              *inputs[6].d[1]),
          *expr_builder.operation(
              nvinfer1::DimensionOperation::kPROD, *two, *inputs[6].d[0]));
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
      return ret;
    } else if (output_index == 3) {
      // pos id
      nvinfer1::DimsExprs ret = inputs[5];
      return ret;
    } else if (output_index == 4) {
      // mask id
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = inputs[6].d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    }
  } else {
    if (output_index == 0) {
      nvinfer1::DimsExprs ret = x_dims;
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = new_mask_dims.d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    }
  }
}

bool FusedTokenPrunePluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (flag_varseqlen_) {
<<<<<<< HEAD
    if (pos <= 3 || pos == 7) {
      if (with_fp16_) {
        return (in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      } else {
        PADDLE_THROW(platform::errors::Fatal(
            "The FusedTokenPrune TRT Plugin's input type "
            "should be half for varseqlen."));
      }
    } else if (pos == 6 || pos == 11) {  // mask_id, mask_id_out
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
=======
    if (pos == 0) {
      if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
        return (in.type == nvinfer1::DataType::kFLOAT ||
                in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
      } else {
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      }
    } else if (pos <= 3 || pos == 7) {
      const nvinfer1::PluginTensorDesc& prev = in_out[0];
      return in.type == prev.type && in.format == prev.format;
    } else if (pos == 6 || pos == 11) {  // mask_id, mask_id_out
      return in.type == nvinfer1::DataType::kFLOAT &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
    } else {
      return in.type == nvinfer1::DataType::kINT32 &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  } else {
    if (pos == 0) {
      if (with_fp16_) {
<<<<<<< HEAD
        return (in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
=======
#ifdef TRT_PLUGIN_FP16_AVALIABLE
        return (in.type == nvinfer1::DataType::kFLOAT ||
                in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
      } else {
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      }
    } else if (pos <= 4) {
      const nvinfer1::PluginTensorDesc& prev = in_out[0];
      return in.type == prev.type && in.format == prev.format;
    } else {
      return in.type == nvinfer1::DataType::kINT32 &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  }
}

nvinfer1::DataType FusedTokenPrunePluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  if (flag_varseqlen_) {
    if (index == 0) {
<<<<<<< HEAD
      return nvinfer1::DataType::kHALF;
    } else if (index == 4) {  // mask id
      return input_types[6];
=======
      return input_types[1];
    } else if (index == 4) {
      return nvinfer1::DataType::kFLOAT;
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
    } else {
      // index = 1,2,3
      return nvinfer1::DataType::kINT32;
    }
  } else {
    if (index == 0) {
      return input_types[1];
    } else {
      // index = 1
      return nvinfer1::DataType::kINT32;
    }
  }
}

size_t FusedTokenPrunePluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs,
    int nb_inputs,
    const nvinfer1::PluginTensorDesc* outputs,
    int nb_outputs) const TRT_NOEXCEPT {
  auto attn_dims = inputs[0].dims;
  auto x_dims = inputs[1].dims;
  auto new_mask_dims = inputs[3].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];

  int slimmed_x_len = new_mask_dims.d[2];
  int total = bsz * nb_head * max_seq_len * max_seq_len;
  size_t size = total * sizeof(float);
  size += bsz * max_seq_len * sizeof(float);
  size += bsz * max_seq_len * sizeof(int32_t);
  size += bsz * max_seq_len * sizeof(float);
  size += bsz * max_seq_len * sizeof(int32_t);
  size += (bsz + 1) * sizeof(int);
  size += bsz * slimmed_x_len * sizeof(int32_t);
  return size;
}

<<<<<<< HEAD
=======
template <typename T>
inline void enqueueImpl(const nvinfer1::PluginTensorDesc* input_desc,
                        const nvinfer1::PluginTensorDesc* output_desc,
                        const void* const* inputs,
                        void* const* outputs,
                        void* workspace_ptr,
                        hipStream_t stream,
                        int device_id,
                        T max_value,
                        bool keep_first_token_,
                        bool keep_order_) {
  // Dims
  auto attn_dims = input_desc[0].dims;
  auto x_dims = input_desc[1].dims;
  auto new_mask_dims = input_desc[3].dims;

  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  auto c = x_dims.d[2];
  auto slimmed_x_len = new_mask_dims.d[2];

  // Inputs
  const T* attn_data = static_cast<const T*>(inputs[0]);
  const T* x_data = static_cast<const T*>(inputs[1]);
  const T* mask_data = static_cast<const T*>(inputs[2]);

  // Outputs
  T* output_data = static_cast<T*>(outputs[0]);
  int32_t* output_indices_data = static_cast<int32_t*>(outputs[1]);

  int total = bsz * nb_head * max_seq_len * max_seq_len;
  int block = operators::ComputeBlockSize(max_seq_len);
  int grid = operators::CeilDivide(total, block);

  // Workspace for intermediate variable
  char* workspace = static_cast<char*>(workspace_ptr);
  T* attn_tmp_data = reinterpret_cast<T*>(workspace);
  size_t offset = total * sizeof(T);
  T* attn_accu_data = reinterpret_cast<T*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(T);
  int32_t* attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(int32_t);
  T* sort_attn_accu_data = reinterpret_cast<T*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(T);
  int32_t* sort_attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(int32_t);
  int* offsets_data = reinterpret_cast<int*>(workspace + offset);
  offset += (bsz + 1) * sizeof(int);
  int32_t* slimmed_sort_attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);

  // 1. Filter attn by mask
  ElementwiseMask<T>
      <<<grid, block, 0, stream>>>(attn_data, mask_data, attn_tmp_data, total);

  total = bsz * max_seq_len;
  block = operators::ComputeBlockSize(max_seq_len);
  grid = operators::CeilDivide(total, block);
  FillZero<T><<<grid, block, 0, stream>>>(attn_accu_data, total);

  // 2. Reduce sum
  total = bsz * nb_head * max_seq_len * max_seq_len;
  int block_tmp = max_seq_len;
  while (block_tmp > 1024)
    block_tmp /= 2;  // if max seq len > 1024, it must be 2^n
  block =
      block_tmp;  // make sure max_seq_len is an integral multiple of block_size
  grid = operators::CeilDivide(total, block);
  ReduceSum2<T><<<grid, block, block * sizeof(T), stream>>>(
      attn_tmp_data, attn_accu_data, bsz, nb_head, max_seq_len);

  // 3. Prepare token indices
  total = bsz * max_seq_len;
  block = operators::ComputeBlockSize(max_seq_len);
  grid = operators::CeilDivide(total, block);

  FillIndex<<<grid, block, 0, stream>>>(
      attn_accu_indices_data, bsz, max_seq_len);

  // 4. Sort token indices by attn
  if (keep_first_token_) {
    MaximumFirst<T>
        <<<bsz, 1, 0, stream>>>(attn_accu_data, bsz, max_seq_len, max_value);
  }
  size_t temp_storage_bytes = -1;
  int num_items = bsz * max_seq_len;
  int num_segments = bsz;
  FillOffsets<<<bsz + 1, 1, 0, stream>>>(offsets_data, bsz, max_seq_len);
  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr,
      temp_storage_bytes,
      attn_accu_data,
      sort_attn_accu_data,
      attn_accu_indices_data,
      sort_attn_accu_indices_data,
      num_items,
      num_segments,
      offsets_data,
      offsets_data + 1,
      0,
      sizeof(T) * 8,
      stream));
  int64_t temp_size = temp_storage_bytes;
  framework::Tensor temp_storage;
  auto* temp_storage_data = temp_storage.mutable_data<uint8_t>(
      {temp_size}, platform::CUDAPlace(device_id));

  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      temp_storage_data,
      temp_storage_bytes,
      attn_accu_data,
      sort_attn_accu_data,
      attn_accu_indices_data,
      sort_attn_accu_indices_data,
      num_items,
      num_segments,
      offsets_data,
      offsets_data + 1,
      0,
      sizeof(T) * 8,
      stream));
  // 5. Slice
  total = bsz * slimmed_x_len;
  block = operators::ComputeBlockSize(slimmed_x_len);
  grid = operators::CeilDivide(total, block);

  Slice<int32_t>
      <<<grid, block, 0, stream>>>(sort_attn_accu_indices_data,
                                   slimmed_sort_attn_accu_indices_data,
                                   bsz,
                                   max_seq_len,
                                   slimmed_x_len);

  if (keep_order_) {
    // 6. reorder
    num_items = bsz * slimmed_x_len;
    FillOffsets<<<bsz + 1, 1, 0, stream>>>(offsets_data, bsz, slimmed_x_len);
    temp_storage_bytes = -1;
    PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
        nullptr,
        temp_storage_bytes,
        slimmed_sort_attn_accu_indices_data,
        output_indices_data,
        num_items,
        num_segments,
        offsets_data,
        offsets_data + 1,
        0,
        sizeof(int32_t) * 8,
        stream));

    temp_size = temp_storage_bytes;
    temp_storage.Resize({temp_size});
    temp_storage_data =
        temp_storage.mutable_data<uint8_t>(platform::CUDAPlace(device_id));
    PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
        temp_storage_data,
        temp_storage_bytes,
        slimmed_sort_attn_accu_indices_data,
        output_indices_data,
        num_items,
        num_segments,
        offsets_data,
        offsets_data + 1,
        0,
        sizeof(int32_t) * 8,
        stream));

    TakeAlongAxis<T><<<grid, block, 0, stream>>>(x_data,
                                                 output_data,
                                                 output_indices_data,
                                                 bsz,
                                                 max_seq_len,
                                                 slimmed_x_len,
                                                 c);
  } else {
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(output_indices_data,
                                          slimmed_sort_attn_accu_indices_data,
                                          bsz * slimmed_x_len * sizeof(int32_t),
                                          hipMemcpyDeviceToDevice));
    TakeAlongAxis<T>
        <<<grid, block, 0, stream>>>(x_data,
                                     output_data,
                                     slimmed_sort_attn_accu_indices_data,
                                     bsz,
                                     max_seq_len,
                                     slimmed_x_len,
                                     c);
  }
}

inline void pos_id_prune(const int32_t* input,
                         int32_t* output,
                         int pos_nums,
                         float scale,
                         hipStream_t stream) {
  pos_id_prune_kernel<<<1, 1, 0, stream>>>(input, output, pos_nums, scale);
}

>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
int FusedTokenPrunePluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    const void* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) TRT_NOEXCEPT {
<<<<<<< HEAD
  if (flag_varseqlen_) {
    if (!(input_desc[0].type == nvinfer1::DataType::kHALF &&
          input_desc[1].type == nvinfer1::DataType::kHALF)) {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Token_prune'type must half for varseqlen"));
    }
    float scale =
        static_cast<float>(input_desc[3].dims.d[2]) / input_desc[2].dims.d[2];
    const int32_t* input5 =
        static_cast<const int32_t*>(inputs[5]);            // pre pos id
    int32_t* output3 = static_cast<int32_t*>(outputs[3]);  // new pos id
    half* output0 = static_cast<half*>(outputs[0]);
    const int32_t B = input_desc[1].dims.d[0];  // batchs
    const int32_t max_sequnce_length =
        input_desc[1].dims.d[1];                     // max sequnce length
    const int32_t length = input_desc[1].dims.d[2];  // hidden size
    const half* scores = static_cast<const half*>(inputs[0]);  // reduce sum
    const half* tokens = static_cast<const half*>(inputs[1]);
    int32_t padding_token_length;
    if (max_sequnce_length <= 64) {
      padding_token_length = 64;
    } else if (max_sequnce_length <= 128) {
      padding_token_length = 128;
    } else if (max_sequnce_length <= 256) {
      padding_token_length = 256;
    } else if (max_sequnce_length <= 384) {
      padding_token_length = 384;
    } else if (max_sequnce_length <= 512) {
      padding_token_length = 512;
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Token_prune'token_length must <= 512"));
    }

    // 1. Compute the token length after pruning.
    compute_token_length<<<1, B, 0, stream>>>(
        input5, pruned_token_lengths_, scale);

    // 2. Padding scores
    fill_index_padding_score<half><<<B, padding_token_length, 0, stream>>>(
        token_index_,
        scores,
        max_sequnce_length,
        static_cast<half*>(padding_scores_));

    // 3. compute new pos id
    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  pruned_token_lengths_,
                                  output3,
                                  B + 1);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  pruned_token_lengths_,
                                  output3,
                                  B + 1);

    // 4. sort scores
    if (padding_token_length == 64) {
      general_topk_pair_sort<half, 32, 2><<<B, 32, 0, stream>>>(
          static_cast<half*>(padding_scores_), token_index_);  // 64
    } else if (padding_token_length == 128) {
      general_topk_pair_sort<half, 32, 4><<<B, 32, 0, stream>>>(
          static_cast<half*>(padding_scores_), token_index_);  // 128
    } else if (padding_token_length == 256) {
      general_topk_pair_sort<half, 64, 4><<<B, 64, 0, stream>>>(
          static_cast<half*>(padding_scores_), token_index_);  // 256
    } else if (padding_token_length == 384) {
      general_topk_pair_sort<half, 96, 4><<<B, 96, 0, stream>>>(
          static_cast<half*>(padding_scores_), token_index_);  // 384
    } else {
      general_topk_pair_sort<half, 128, 4><<<B, 128, 0, stream>>>(
          static_cast<half*>(padding_scores_), token_index_);  // 512
    }

    // 5. compute output
    int32_t num_threads;
    if (length < 1024) {
      num_threads = length;
    } else {
      if (length % 512 == 0) {
        num_threads = 512;
      } else if (length % 256 == 0) {
        num_threads = 256;
      } else if (length % 128 == 0) {
        num_threads = 128;
      } else if (length % 64 == 0) {
        num_threads = 64;
      } else if (length % 32 == 0) {
        num_threads = 32;
      } else if (length % 16 == 0) {
        num_threads = 16;
      } else if (length % 8 == 0) {
        num_threads = 8;
      } else if (length % 4 == 0) {
        num_threads = 4;
      } else if (length % 2 == 0) {
        num_threads = 2;
      } else {
        num_threads = 1;
      }
    }
    const dim3 num_blocks(
        B,
        max_sequnce_length,
        length / num_threads);  //  batchs, max_sequnce_length, vector_ength/***
    varlen_prune_token_change_order<<<num_blocks, num_threads, 0, stream>>>(
        tokens, output3, padding_token_length, token_index_, output0);
  } else {
    auto input_type = input_desc[0].type;
    const int32_t B = input_desc[1].dims.d[0];  // batchs
    const int32_t pre_sequnce_length = input_desc[1].dims.d[1];
    const int32_t new_sequnce_length = input_desc[3].dims.d[2];  // new mask
    const int32_t length = input_desc[1].dims.d[2];              // hidden size
    if (input_type == nvinfer1::DataType::kFLOAT) {
      VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp32";
      const float* scores = static_cast<const float*>(inputs[0]);  // reduce sum
      const float* tokens = static_cast<const float*>(inputs[1]);  // X
      float* output0 = static_cast<float*>(outputs[0]);
      int32_t* output1 = static_cast<int32_t*>(outputs[1]);
      int32_t padding_token_length;
      if (pre_sequnce_length <= 64) {
        padding_token_length = 64;
      } else if (pre_sequnce_length <= 128) {
        padding_token_length = 128;
      } else if (pre_sequnce_length <= 256) {
        padding_token_length = 256;
      } else if (pre_sequnce_length <= 384) {
        padding_token_length = 384;
      } else if (pre_sequnce_length <= 512) {
        padding_token_length = 512;
      } else {
        PADDLE_THROW(platform::errors::InvalidArgument(
            "Token_prune'token_length must <= 512"));
      }

      // 1. Padding scores
      fill_index_padding_score<float><<<B, padding_token_length, 0, stream>>>(
          token_index_,
          scores,
          pre_sequnce_length,
          static_cast<float*>(padding_scores_));

      // 2. sort scores
      if (padding_token_length == 64) {
        general_topk_pair_sort<float, 32, 2><<<B, 32, 0, stream>>>(
            static_cast<float*>(padding_scores_), token_index_);  // 64
      } else if (padding_token_length == 128) {
        general_topk_pair_sort<float, 32, 4><<<B, 32, 0, stream>>>(
            static_cast<float*>(padding_scores_), token_index_);  // 128
      } else if (padding_token_length == 256) {
        general_topk_pair_sort<float, 64, 4><<<B, 64, 0, stream>>>(
            static_cast<float*>(padding_scores_), token_index_);  // 256
      } else if (padding_token_length == 384) {
        general_topk_pair_sort<float, 96, 4><<<B, 96, 0, stream>>>(
            static_cast<float*>(padding_scores_), token_index_);  // 384
      } else {
        general_topk_pair_sort<float, 128, 4><<<B, 128, 0, stream>>>(
            static_cast<float*>(padding_scores_), token_index_);  // 512
      }

      // 3. compute output
      int32_t num_threads;
      if (length < 1024) {
        num_threads = length;
      } else {
        if (length % 512 == 0) {
          num_threads = 512;
        } else if (length % 256 == 0) {
          num_threads = 256;
        } else if (length % 128 == 0) {
          num_threads = 128;
        } else if (length % 64 == 0) {
          num_threads = 64;
        } else if (length % 32 == 0) {
          num_threads = 32;
        } else if (length % 16 == 0) {
          num_threads = 16;
        } else if (length % 8 == 0) {
          num_threads = 8;
        } else if (length % 4 == 0) {
          num_threads = 4;
        } else if (length % 2 == 0) {
          num_threads = 2;
        } else {
          num_threads = 1;
        }
      }
      if (keep_order_) {
        const dim3 num_blocks(B, length / num_threads);
        prune_token_keep_order<float>
            <<<num_blocks, num_threads, 0, stream>>>(tokens,
                                                     pre_sequnce_length,
                                                     new_sequnce_length,
                                                     padding_token_length,
                                                     token_index_,
                                                     output0,
                                                     output1);
      } else {
        const dim3 num_blocks(B, pre_sequnce_length, length / num_threads);
        prune_token_change_order<float>
            <<<num_blocks, num_threads, 0, stream>>>(tokens,
                                                     new_sequnce_length,
                                                     padding_token_length,
                                                     token_index_,
                                                     output0);
      }
    } else if (input_type == nvinfer1::DataType::kHALF) {
      VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp16";
      const half* scores = static_cast<const half*>(inputs[0]);  // reduce sum
      const half* tokens = static_cast<const half*>(inputs[1]);  // X
      half* output0 = static_cast<half*>(outputs[0]);
      int32_t* output1 = static_cast<int32_t*>(outputs[1]);
      int32_t padding_token_length;
      if (pre_sequnce_length <= 64) {
        padding_token_length = 64;
      } else if (pre_sequnce_length <= 128) {
        padding_token_length = 128;
      } else if (pre_sequnce_length <= 256) {
        padding_token_length = 256;
      } else if (pre_sequnce_length <= 384) {
        padding_token_length = 384;
      } else if (pre_sequnce_length <= 512) {
        padding_token_length = 512;
      } else {
        PADDLE_THROW(platform::errors::InvalidArgument(
            "Token_prune'token_length must <= 512"));
      }

      // 1. Padding scores
      fill_index_padding_score<half><<<B, padding_token_length, 0, stream>>>(
          token_index_,
          scores,
          pre_sequnce_length,
          static_cast<half*>(padding_scores_));

      // 2. sort scores
      if (padding_token_length == 64) {
        general_topk_pair_sort<half, 32, 2><<<B, 32, 0, stream>>>(
            static_cast<half*>(padding_scores_), token_index_);  // 64
      } else if (padding_token_length == 128) {
        general_topk_pair_sort<half, 32, 4><<<B, 32, 0, stream>>>(
            static_cast<half*>(padding_scores_), token_index_);  // 128
      } else if (padding_token_length == 256) {
        general_topk_pair_sort<half, 64, 4><<<B, 64, 0, stream>>>(
            static_cast<half*>(padding_scores_), token_index_);  // 256
      } else if (padding_token_length == 384) {
        general_topk_pair_sort<half, 96, 4><<<B, 96, 0, stream>>>(
            static_cast<half*>(padding_scores_), token_index_);  // 384
      } else {
        general_topk_pair_sort<half, 128, 4><<<B, 128, 0, stream>>>(
            static_cast<half*>(padding_scores_), token_index_);  // 512
      }

      // 3. compute output
      int32_t num_threads;
      if (length < 1024) {
        num_threads = length;
      } else {
        if (length % 512 == 0) {
          num_threads = 512;
        } else if (length % 256 == 0) {
          num_threads = 256;
        } else if (length % 128 == 0) {
          num_threads = 128;
        } else if (length % 64 == 0) {
          num_threads = 64;
        } else if (length % 32 == 0) {
          num_threads = 32;
        } else if (length % 16 == 0) {
          num_threads = 16;
        } else if (length % 8 == 0) {
          num_threads = 8;
        } else if (length % 4 == 0) {
          num_threads = 4;
        } else if (length % 2 == 0) {
          num_threads = 2;
        } else {
          num_threads = 1;
        }
      }
      if (keep_order_) {
        const dim3 num_blocks(B, length / num_threads);
        prune_token_keep_order<half>
            <<<num_blocks, num_threads, 0, stream>>>(tokens,
                                                     pre_sequnce_length,
                                                     new_sequnce_length,
                                                     padding_token_length,
                                                     token_index_,
                                                     output0,
                                                     output1);
      } else {
        const dim3 num_blocks(B, pre_sequnce_length, length / num_threads);
        prune_token_change_order<half>
            <<<num_blocks, num_threads, 0, stream>>>(tokens,
                                                     new_sequnce_length,
                                                     padding_token_length,
                                                     token_index_,
                                                     output0);
      }
    } else {
      PADDLE_THROW(
          platform::errors::Fatal("The FusedTokenPrune TRT Plugin's input type "
                                  "should be float or half."));
    }
=======
  auto input_type = input_desc[0].type;
  auto attn_dims = input_desc[0].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  int device_id;
  hipGetDevice(&device_id);

  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp32";

    float max = std::numeric_limits<float>::max();

    enqueueImpl<float>(input_desc,
                       output_desc,
                       inputs,
                       outputs,
                       workspace,
                       stream,
                       device_id,
                       max,
                       keep_first_token_,
                       keep_order_);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp16";

    half max = 65504.0;

    enqueueImpl<half>(input_desc,
                      output_desc,
                      inputs,
                      outputs,
                      workspace,
                      stream,
                      device_id,
                      max,
                      keep_first_token_,
                      keep_order_);

#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) TensorRT Plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The FusedTokenPrune TRT Plugin's input type "
                                "should be float or half."));
  }
  if (flag_varseqlen_) {
    float scale =
        static_cast<float>(input_desc[3].dims.d[2]) / input_desc[6].dims.d[1];
    // outputs[2]=inputs[4]; // word_id
    const int32_t* inputs5 = static_cast<const int32_t*>(inputs[5]);
    int32_t* outputs3 = static_cast<int32_t*>(outputs[3]);
    pos_id_prune(
        inputs5, outputs3, input_desc[5].dims.d[0], scale, stream);  // pos_id
    // outputs[4]=inputs[6]; // new_mask
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
  }
  return hipGetLastError() != hipSuccess;
}

<<<<<<< HEAD
=======
#endif
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
