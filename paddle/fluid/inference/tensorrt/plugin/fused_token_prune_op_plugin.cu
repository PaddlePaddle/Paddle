#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>

#include "hipcub/hipcub.hpp"

#include "paddle/phi/kernels/funcs/math_function.h"

#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/device_context.h"

#include "paddle/fluid/inference/tensorrt/plugin/fused_token_prune_op_plugin.h"
#include "paddle/fluid/operators/fused_token_prune_op.cu.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

#if IS_TRT_VERSION_GE(6000)

template <typename T>
__global__ void ElementwiseMask(const T* a,
                                const T* b,
                                T* res,
                                int num_elements) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_elements) return;
  const T zero = 0;
  res[tid] = b[tid] >= zero ? a[tid] : zero;
#endif
}

template <typename T>
__global__ void FillZero(T* data, int len) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  const T zero = 0;
  data[tid] = zero;
}

__global__ void FillIndex(int32_t* indices, int num_raws, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * num_cols) return;

  int col = tid % num_cols;
  int raw = tid / num_cols;

  indices[tid] = col;
}

template <typename T>
__global__ void MaximumFirst(T* mat, int num_raws, int num_cols, T max_value) {
  auto raw = blockIdx.x * blockDim.x + threadIdx.x;
  if (raw >= num_raws) return;
  mat[raw * num_cols] = max_value;
}

__global__ void FillOffsets(int* offsets, int num_raws, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid > num_raws) return;

  offsets[tid] = tid * num_cols;
}

template <typename T>
__global__ void Slice(
    const T* src, T* dst, int num_raws, int src_num_cols, int dst_num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * dst_num_cols) return;
  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  dst[tid] = src[raw * src_num_cols + col];
}

template <typename T>
__global__ void ReduceSum2(
    const T* src, T* dst, int bsz, int nb_head, int max_seq_len) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_blocks_per_head = ((max_seq_len / blockDim.x) * max_seq_len);
  int batch = bid / (nb_head * num_blocks_per_head);
  int col = bid % max_seq_len;
  int head = (bid / num_blocks_per_head) % nb_head;

  extern __shared__ T res_float[];
  res_float[tid] =
      src[batch * (nb_head * max_seq_len * max_seq_len) +
          head * (max_seq_len * max_seq_len) + col + tid * max_seq_len];
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      res_float[tid] += res_float[tid + offset];
    }
    __syncthreads();
    if (offset % 2 == 1 && tid == offset - 2) {
      res_float[tid] += res_float[tid + 1];
    }
  }

  if (tid == 0) {
    auto* dst_addr = dst + batch * max_seq_len + col;
    atomicAdd(dst_addr, res_float[0]);
  }
}

template <>
__global__ void ReduceSum2<half>(
    const half* src, half* dst, int bsz, int nb_head, int max_seq_len) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_blocks_per_head = ((max_seq_len / blockDim.x) * max_seq_len);
  int batch = bid / (nb_head * num_blocks_per_head);
  int col = bid % max_seq_len;
  int head = (bid / num_blocks_per_head) % nb_head;

  extern __shared__ half res_half[];
  res_half[tid] =
      src[batch * (nb_head * max_seq_len * max_seq_len) +
          head * (max_seq_len * max_seq_len) + col + tid * max_seq_len];
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      res_half[tid] += res_half[tid + offset];
    }
    __syncthreads();
    if (offset % 2 == 1 && tid == offset - 2) {
      res_half[tid] += res_half[tid + 1];
    }
    __syncthreads();
  }

  if (tid == 0) {
    platform::fastAtomicAdd<platform::float16>(
        reinterpret_cast<platform::float16*>(dst),
        static_cast<size_t>(batch * max_seq_len + col),
        static_cast<size_t>(bsz * max_seq_len),
        static_cast<platform::float16>(res_half[0]));
  }
#endif
}

template <typename T>
__global__ void TakeAlongAxis(const T* src,
                              T* dst,
                              int32_t* indices,
                              int num_raws,
                              int src_num_cols,
                              int dst_num_cols,
                              int num_elements) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * dst_num_cols) return;

  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  for (int i = 0; i < num_elements; ++i) {
    dst[tid * num_elements + i] =
        *(src + (raw * src_num_cols + indices[tid]) * num_elements + i);
  }
}

__global__ void pos_id_prune_kernel(const int32_t* src,
                                    int32_t* dst,
                                    int pos_nums,
                                    float scale) {
  dst[0] = 0;
  for (int i = 1; i < pos_nums; i++) {
    dst[i] =
        dst[i - 1] + max(static_cast<int>((src[i] - src[i - 1]) * scale), 2);
  }
}

nvinfer1::DimsExprs FusedTokenPrunePluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs* inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  auto x_dims = inputs[1], new_mask_dims = inputs[3];
  if (flag_varseqlen_) {
    if (output_index == 0) {
      nvinfer1::DimsExprs ret = x_dims;
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else if (output_index == 1) {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = new_mask_dims.d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else if (output_index == 2) {
      // word id
      nvinfer1::DimsExprs ret;
      ret.nbDims = 1;
      // max sum of seqlen: pre_seqlen * new_mask[2] / mask[1] + 2 * batchs
      const auto* two = expr_builder.constant(2);
      ret.d[0] = expr_builder.operation(
          nvinfer1::DimensionOperation::kSUM,
          *expr_builder.operation(
              nvinfer1::DimensionOperation::kFLOOR_DIV,
              *expr_builder.operation(nvinfer1::DimensionOperation::kPROD,
                                      *inputs[4].d[0],
                                      *new_mask_dims.d[2]),
              *inputs[6].d[1]),
          *expr_builder.operation(
              nvinfer1::DimensionOperation::kPROD, *two, *inputs[6].d[0]));
      return ret;
    } else if (output_index == 3) {
      // pos id
      nvinfer1::DimsExprs ret = inputs[5];
      return ret;
    } else if (output_index == 4) {
      // mask id
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = inputs[6].d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    }
  } else {
    if (output_index == 0) {
      nvinfer1::DimsExprs ret = x_dims;
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = new_mask_dims.d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    }
  }
}

bool FusedTokenPrunePluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (flag_varseqlen_) {
    if (pos == 0) {
      if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
        return (in.type == nvinfer1::DataType::kFLOAT ||
                in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
      } else {
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      }
    } else if (pos <= 3 || pos == 7) {
      const nvinfer1::PluginTensorDesc& prev = in_out[0];
      return in.type == prev.type && in.format == prev.format;
    } else if (pos == 6 || pos == 11) {  // mask_id, mask_id_out
      return in.type == nvinfer1::DataType::kFLOAT &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    } else {
      return in.type == nvinfer1::DataType::kINT32 &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  } else {
    if (pos == 0) {
      if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
        return (in.type == nvinfer1::DataType::kFLOAT ||
                in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
      } else {
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      }
    } else if (pos <= 4) {
      const nvinfer1::PluginTensorDesc& prev = in_out[0];
      return in.type == prev.type && in.format == prev.format;
    } else {
      return in.type == nvinfer1::DataType::kINT32 &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  }
}

nvinfer1::DataType FusedTokenPrunePluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  if (flag_varseqlen_) {
    if (index == 0) {
      return input_types[1];
    } else if (index == 4) {
      return nvinfer1::DataType::kFLOAT;
    } else {
      // index = 1,2,3
      return nvinfer1::DataType::kINT32;
    }
  } else {
    if (index == 0) {
      return input_types[1];
    } else {
      // index = 1
      return nvinfer1::DataType::kINT32;
    }
  }
}

size_t FusedTokenPrunePluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs,
    int nb_inputs,
    const nvinfer1::PluginTensorDesc* outputs,
    int nb_outputs) const TRT_NOEXCEPT {
  auto attn_dims = inputs[0].dims;
  auto x_dims = inputs[1].dims;
  auto new_mask_dims = inputs[3].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];

  int slimmed_x_len = new_mask_dims.d[2];
  int total = bsz * nb_head * max_seq_len * max_seq_len;
  size_t size = total * sizeof(float);
  size += bsz * max_seq_len * sizeof(float);
  size += bsz * max_seq_len * sizeof(int32_t);
  size += bsz * max_seq_len * sizeof(float);
  size += bsz * max_seq_len * sizeof(int32_t);
  size += (bsz + 1) * sizeof(int);
  size += bsz * slimmed_x_len * sizeof(int32_t);
  return size;
}

template <typename T>
inline void enqueueImpl(const nvinfer1::PluginTensorDesc* input_desc,
                        const nvinfer1::PluginTensorDesc* output_desc,
                        const void* const* inputs,
                        void* const* outputs,
                        void* workspace_ptr,
                        hipStream_t stream,
                        int device_id,
                        T max_value,
                        bool keep_first_token_,
                        bool keep_order_) {
  // Dims
  auto attn_dims = input_desc[0].dims;
  auto x_dims = input_desc[1].dims;
  auto new_mask_dims = input_desc[3].dims;

  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  auto c = x_dims.d[2];
  auto slimmed_x_len = new_mask_dims.d[2];

  // Inputs
  const T* attn_data = static_cast<const T*>(inputs[0]);
  const T* x_data = static_cast<const T*>(inputs[1]);
  const T* mask_data = static_cast<const T*>(inputs[2]);

  // Outputs
  T* output_data = static_cast<T*>(outputs[0]);
  int32_t* output_indices_data = static_cast<int32_t*>(outputs[1]);

  int total = bsz * nb_head * max_seq_len * max_seq_len;
  int block = operators::ComputeBlockSize(max_seq_len);
  int grid = operators::CeilDivide(total, block);

  // Workspace for intermediate variable
  char* workspace = static_cast<char*>(workspace_ptr);
  T* attn_tmp_data = reinterpret_cast<T*>(workspace);
  size_t offset = total * sizeof(T);
  T* attn_accu_data = reinterpret_cast<T*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(T);
  int32_t* attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(int32_t);
  T* sort_attn_accu_data = reinterpret_cast<T*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(T);
  int32_t* sort_attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(int32_t);
  int* offsets_data = reinterpret_cast<int*>(workspace + offset);
  offset += (bsz + 1) * sizeof(int);
  int32_t* slimmed_sort_attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);

  // 1. Filter attn by mask
  ElementwiseMask<T>
      <<<grid, block, 0, stream>>>(attn_data, mask_data, attn_tmp_data, total);

  total = bsz * max_seq_len;
  block = operators::ComputeBlockSize(max_seq_len);
  grid = operators::CeilDivide(total, block);
  FillZero<T><<<grid, block, 0, stream>>>(attn_accu_data, total);

  // 2. Reduce sum
  total = bsz * nb_head * max_seq_len * max_seq_len;
  int block_tmp = max_seq_len;
  while (block_tmp > 1024)
    block_tmp /= 2;  // if max seq len > 1024, it must be 2^n
  block =
      block_tmp;  // make sure max_seq_len is an integral multiple of block_size
  grid = operators::CeilDivide(total, block);
  ReduceSum2<T><<<grid, block, block * sizeof(T), stream>>>(
      attn_tmp_data, attn_accu_data, bsz, nb_head, max_seq_len);

  // 3. Prepare token indices
  total = bsz * max_seq_len;
  block = operators::ComputeBlockSize(max_seq_len);
  grid = operators::CeilDivide(total, block);

  FillIndex<<<grid, block, 0, stream>>>(
      attn_accu_indices_data, bsz, max_seq_len);

  // 4. Sort token indices by attn
  if (keep_first_token_) {
    MaximumFirst<T>
        <<<bsz, 1, 0, stream>>>(attn_accu_data, bsz, max_seq_len, max_value);
  }
  size_t temp_storage_bytes = -1;
  int num_items = bsz * max_seq_len;
  int num_segments = bsz;
  FillOffsets<<<bsz + 1, 1, 0, stream>>>(offsets_data, bsz, max_seq_len);
  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr,
      temp_storage_bytes,
      attn_accu_data,
      sort_attn_accu_data,
      attn_accu_indices_data,
      sort_attn_accu_indices_data,
      num_items,
      num_segments,
      offsets_data,
      offsets_data + 1,
      0,
      sizeof(T) * 8,
      stream));
  int64_t temp_size = temp_storage_bytes;
  framework::Tensor temp_storage;
  auto* temp_storage_data = temp_storage.mutable_data<uint8_t>(
      {temp_size}, platform::CUDAPlace(device_id));

  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      temp_storage_data,
      temp_storage_bytes,
      attn_accu_data,
      sort_attn_accu_data,
      attn_accu_indices_data,
      sort_attn_accu_indices_data,
      num_items,
      num_segments,
      offsets_data,
      offsets_data + 1,
      0,
      sizeof(T) * 8,
      stream));
  // 5. Slice
  total = bsz * slimmed_x_len;
  block = operators::ComputeBlockSize(slimmed_x_len);
  grid = operators::CeilDivide(total, block);

  Slice<int32_t>
      <<<grid, block, 0, stream>>>(sort_attn_accu_indices_data,
                                   slimmed_sort_attn_accu_indices_data,
                                   bsz,
                                   max_seq_len,
                                   slimmed_x_len);

  if (keep_order_) {
    // 6. reorder
    num_items = bsz * slimmed_x_len;
    FillOffsets<<<bsz + 1, 1, 0, stream>>>(offsets_data, bsz, slimmed_x_len);
    temp_storage_bytes = -1;
    PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
        nullptr,
        temp_storage_bytes,
        slimmed_sort_attn_accu_indices_data,
        output_indices_data,
        num_items,
        num_segments,
        offsets_data,
        offsets_data + 1,
        0,
        sizeof(int32_t) * 8,
        stream));

    temp_size = temp_storage_bytes;
    temp_storage.Resize({temp_size});
    temp_storage_data =
        temp_storage.mutable_data<uint8_t>(platform::CUDAPlace(device_id));
    PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
        temp_storage_data,
        temp_storage_bytes,
        slimmed_sort_attn_accu_indices_data,
        output_indices_data,
        num_items,
        num_segments,
        offsets_data,
        offsets_data + 1,
        0,
        sizeof(int32_t) * 8,
        stream));

    TakeAlongAxis<T><<<grid, block, 0, stream>>>(x_data,
                                                 output_data,
                                                 output_indices_data,
                                                 bsz,
                                                 max_seq_len,
                                                 slimmed_x_len,
                                                 c);
  } else {
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(output_indices_data,
                                          slimmed_sort_attn_accu_indices_data,
                                          bsz * slimmed_x_len * sizeof(int32_t),
                                          hipMemcpyDeviceToDevice));
    TakeAlongAxis<T>
        <<<grid, block, 0, stream>>>(x_data,
                                     output_data,
                                     slimmed_sort_attn_accu_indices_data,
                                     bsz,
                                     max_seq_len,
                                     slimmed_x_len,
                                     c);
  }
}

inline void pos_id_prune(const int32_t* input,
                         int32_t* output,
                         int pos_nums,
                         float scale,
                         hipStream_t stream) {
  pos_id_prune_kernel<<<1, 1, 0, stream>>>(input, output, pos_nums, scale);
}

int FusedTokenPrunePluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    const void* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  auto input_type = input_desc[0].type;
  auto attn_dims = input_desc[0].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  int device_id;
  hipGetDevice(&device_id);

  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp32";

    float max = std::numeric_limits<float>::max();

    enqueueImpl<float>(input_desc,
                       output_desc,
                       inputs,
                       outputs,
                       workspace,
                       stream,
                       device_id,
                       max,
                       keep_first_token_,
                       keep_order_);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp16";

    half max = 65504.0;

    enqueueImpl<half>(input_desc,
                      output_desc,
                      inputs,
                      outputs,
                      workspace,
                      stream,
                      device_id,
                      max,
                      keep_first_token_,
                      keep_order_);

#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) TensorRT Plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The FusedTokenPrune TRT Plugin's input type "
                                "should be float or half."));
  }
  if (flag_varseqlen_) {
    float scale =
        static_cast<float>(input_desc[3].dims.d[2]) / input_desc[6].dims.d[1];
    // outputs[2]=inputs[4]; // word_id
    const int32_t* inputs5 = static_cast<const int32_t*>(inputs[5]);
    int32_t* outputs3 = static_cast<int32_t*>(outputs[3]);
    pos_id_prune(
        inputs5, outputs3, input_desc[5].dims.d[0], scale, stream);  // pos_id
    // outputs[4]=inputs[6]; // new_mask
  }
  return hipGetLastError() != hipSuccess;
}

#endif
}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
