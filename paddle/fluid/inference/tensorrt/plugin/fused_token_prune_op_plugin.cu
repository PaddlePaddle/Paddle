#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>

#include "hipcub/hipcub.hpp"

#include "paddle/phi/kernels/funcs/math_function.h"

#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"

#include "paddle/fluid/inference/tensorrt/plugin/fused_token_prune_op_plugin.h"
#include "paddle/fluid/operators/fused_token_prune_op.cu.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

template <typename T>
__global__ void ElementwiseMask(const T* a,
                                const T* b,
                                T* res,
                                int num_elements) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_elements) return;
  const T zero = 0;
  res[tid] = b[tid] >= zero ? a[tid] : zero;
#endif
}

template <typename T>
__global__ void FillZero(T* data, int len) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  const T zero = 0;
  data[tid] = zero;
}

__global__ void FillIndex(int32_t* indices, int num_raws, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * num_cols) return;

  int col = tid % num_cols;
  int raw = tid / num_cols;

  indices[tid] = col;
}

template <typename T>
__global__ void MaximumFirst(T* mat, int num_raws, int num_cols, T max_value) {
  auto raw = blockIdx.x * blockDim.x + threadIdx.x;
  if (raw >= num_raws) return;
  mat[raw * num_cols] = max_value;
}

__global__ void FillOffsets(int* offsets, int num_raws, int num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid > num_raws) return;

  offsets[tid] = tid * num_cols;
}

template <typename T>
__global__ void Slice(
    const T* src, T* dst, int num_raws, int src_num_cols, int dst_num_cols) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * dst_num_cols) return;
  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  dst[tid] = src[raw * src_num_cols + col];
}

template <typename T>
__global__ void ReduceSum2(
    const T* src, T* dst, int bsz, int nb_head, int max_seq_len) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_blocks_per_head = ((max_seq_len / blockDim.x) * max_seq_len);
  int batch = bid / (nb_head * num_blocks_per_head);
  int col = bid % max_seq_len;
  int head = (bid / num_blocks_per_head) % nb_head;

  extern __shared__ T res_float[];
  res_float[tid] =
      src[batch * (nb_head * max_seq_len * max_seq_len) +
          head * (max_seq_len * max_seq_len) + col + tid * max_seq_len];
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      res_float[tid] += res_float[tid + offset];
    }
    __syncthreads();
    if (offset % 2 == 1 && tid == offset - 2) {
      res_float[tid] += res_float[tid + 1];
    }
  }

  if (tid == 0) {
    auto* dst_addr = dst + batch * max_seq_len + col;
    atomicAdd(dst_addr, res_float[0]);
  }
}

template <>
__global__ void ReduceSum2<half>(
    const half* src, half* dst, int bsz, int nb_head, int max_seq_len) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int num_blocks_per_head = ((max_seq_len / blockDim.x) * max_seq_len);
  int batch = bid / (nb_head * num_blocks_per_head);
  int col = bid % max_seq_len;
  int head = (bid / num_blocks_per_head) % nb_head;

  extern __shared__ half res_half[];
  res_half[tid] =
      src[batch * (nb_head * max_seq_len * max_seq_len) +
          head * (max_seq_len * max_seq_len) + col + tid * max_seq_len];
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      res_half[tid] += res_half[tid + offset];
    }
    __syncthreads();
    if (offset % 2 == 1 && tid == offset - 2) {
      res_half[tid] += res_half[tid + 1];
    }
    __syncthreads();
  }

  if (tid == 0) {
    phi::fastAtomicAdd<platform::float16>(
        reinterpret_cast<platform::float16*>(dst),
        static_cast<size_t>(batch * max_seq_len + col),
        static_cast<size_t>(bsz * max_seq_len),
        static_cast<platform::float16>(res_half[0]));
  }
#endif
}

template <typename T>
__global__ void TakeAlongAxis(const T* src,
                              T* dst,
                              int32_t* indices,
                              int num_raws,
                              int src_num_cols,
                              int dst_num_cols,
                              int num_elements) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= num_raws * dst_num_cols) return;

  int raw = tid / dst_num_cols;
  int col = tid % dst_num_cols;
  for (int i = 0; i < num_elements; ++i) {
    dst[tid * num_elements + i] =
        *(src + (raw * src_num_cols + indices[tid]) * num_elements + i);
  }
}

__global__ void compute_token_length(const int32_t* src,
                                     int32_t* dst,
                                     float scale) {
  int32_t it = threadIdx.x;
  dst[it] = max(static_cast<int>((src[it + 1] - src[it]) * scale), 1);
}

__global__ void fill_index_padding_score(int32_t* token_index,
                                         const half* scores,
                                         int32_t scores_size,
                                         half* padding_scores) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  token_index[tid] = threadIdx.x;
  if (tid < scores_size) {
    padding_scores[tid] = scores[tid];
  } else {
    padding_scores[tid] = 0;
  }
}

template <typename T, int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void general_topk_pair_sort(T* in_keys, int32_t* in_out_values) {
  typedef hipcub::BlockRadixSort<T, BLOCK_THREADS, ITEMS_PER_THREAD, int>
      BlockRadixSort;
  typedef cub::
      BlockLoad<T, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE>
          BlockLoadKey;
  typedef cub::
      BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE>
          BlockLoadValue;
  typedef cub::
      BlockStore<T, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE>
          BlockStoreKey;
  typedef hipcub::BlockStore<int,
                          BLOCK_THREADS,
                          ITEMS_PER_THREAD,
                          hipcub::BLOCK_STORE_TRANSPOSE>
      BlockStoreValue;

  __shared__ union {
    typename BlockRadixSort::TempStorage sort;
    typename BlockLoadKey::TempStorage loadkey;
    typename BlockLoadValue::TempStorage loadvalue;
    typename BlockStoreKey::TempStorage storekey;
    typename BlockStoreValue::TempStorage storevalue;
  } temp_storage;

  int block_offset = blockIdx.x * BLOCK_THREADS * ITEMS_PER_THREAD;

  T thread_keys[ITEMS_PER_THREAD];
  int thread_values[ITEMS_PER_THREAD];
  BlockLoadKey(temp_storage.loadkey).Load(in_keys + block_offset, thread_keys);
  BlockLoadValue(temp_storage.loadvalue)
      .Load(in_out_values + block_offset, thread_values);
  __syncthreads();

  BlockRadixSort(temp_storage.sort).SortDescending(thread_keys, thread_values);
  __syncthreads();

  BlockStoreValue(temp_storage.storevalue)
      .Store(in_out_values + block_offset, thread_values);
}

__global__ void varlen_prune_token(const half* tokens,
                                   const int32_t* token_pos,
                                   const int32_t* token_index,
                                   half* output) {
  int batch = blockIdx.x;
  int token_it = batch * gridDim.y + blockIdx.y;
  int pre_value_it =
      token_it * gridDim.z * blockDim.x + blockIdx.z * blockDim.x + threadIdx.x;

  if (token_index[token_it] < token_pos[batch + 1] - token_pos[batch]) {
    output[(token_index[token_it] + token_pos[batch]) * gridDim.z * blockDim.x +
           blockIdx.z * blockDim.x + threadIdx.x] = tokens[pre_value_it];
  }
}

nvinfer1::DimsExprs FusedTokenPrunePluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs* inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  auto x_dims = inputs[1], new_mask_dims = inputs[3];
  if (flag_varseqlen_) {
    // max sum of seqlen: ceil(sum / scale) + n -1 >= for(i=0;i<n;i++) {sum +=
    // floor(num(i) / scale)} auto
    // pruned_sum_length=std::ceil(inputs[4].d[0]*new_mask_dims.d[2]/inputs[6].d[1])+
    // inputs[1].d[0] - 1;
    auto pruned_sum_length = expr_builder.operation(
        nvinfer1::DimensionOperation::kSUB,
        *expr_builder.operation(
            nvinfer1::DimensionOperation::kSUM,
            *expr_builder.operation(
                nvinfer1::DimensionOperation::kCEIL_DIV,
                *expr_builder.operation(nvinfer1::DimensionOperation::kPROD,
                                        *inputs[4].d[0],
                                        *new_mask_dims.d[2]),
                *inputs[6].d[1]),
            *inputs[1].d[0]),
        *expr_builder.constant(1));
    if (output_index == 0) {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 4;
      ret.d[0] = pruned_sum_length;
      ret.d[1] = x_dims.d[2];
      ret.d[2] = expr_builder.constant(1);
      ret.d[3] = expr_builder.constant(1);
      return ret;
    } else if (output_index == 1) {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = new_mask_dims.d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else if (output_index == 2) {
      // word id
      nvinfer1::DimsExprs ret;
      ret.nbDims = 1;
      ret.d[0] = pruned_sum_length;
      return ret;
    } else if (output_index == 3) {
      // pos id
      nvinfer1::DimsExprs ret = inputs[5];
      return ret;
    } else if (output_index == 4) {
      // mask id
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = inputs[6].d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    }
  } else {
    if (output_index == 0) {
      nvinfer1::DimsExprs ret = x_dims;
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    } else {
      nvinfer1::DimsExprs ret;
      ret.nbDims = 2;
      ret.d[0] = new_mask_dims.d[0];
      ret.d[1] = new_mask_dims.d[2];
      return ret;
    }
  }
}

bool FusedTokenPrunePluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc* in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (flag_varseqlen_) {
    if (pos <= 3 || pos == 7) {
      if (with_fp16_) {
        return (in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      } else {
        PADDLE_THROW(platform::errors::Fatal(
            "The FusedTokenPrune TRT Plugin's input type "
            "should be half for varseqlen."));
      }
    } else if (pos == 6 || pos == 11) {  // mask_id, mask_id_out
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    } else {
      return in.type == nvinfer1::DataType::kINT32 &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  } else {
    if (pos == 0) {
      if (with_fp16_) {
        return (in.type == nvinfer1::DataType::kHALF) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);

      } else {
        return (in.type == nvinfer1::DataType::kFLOAT) &&
               (in.format == nvinfer1::TensorFormat::kLINEAR);
      }
    } else if (pos <= 4) {
      const nvinfer1::PluginTensorDesc& prev = in_out[0];
      return in.type == prev.type && in.format == prev.format;
    } else {
      return in.type == nvinfer1::DataType::kINT32 &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  }
}

nvinfer1::DataType FusedTokenPrunePluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  if (flag_varseqlen_) {
    if (index == 0) {
      return nvinfer1::DataType::kHALF;
    } else if (index == 4) {  // mask id
      return input_types[6];
    } else {
      // index = 1,2,3
      return nvinfer1::DataType::kINT32;
    }
  } else {
    if (index == 0) {
      return input_types[1];
    } else {
      // index = 1
      return nvinfer1::DataType::kINT32;
    }
  }
}

size_t FusedTokenPrunePluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs,
    int nb_inputs,
    const nvinfer1::PluginTensorDesc* outputs,
    int nb_outputs) const TRT_NOEXCEPT {
  auto attn_dims = inputs[0].dims;
  auto x_dims = inputs[1].dims;
  auto new_mask_dims = inputs[3].dims;
  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];

  int slimmed_x_len = new_mask_dims.d[2];
  int total = bsz * nb_head * max_seq_len * max_seq_len;
  size_t size = total * sizeof(float);
  size += bsz * max_seq_len * sizeof(float);
  size += bsz * max_seq_len * sizeof(int32_t);
  size += bsz * max_seq_len * sizeof(float);
  size += bsz * max_seq_len * sizeof(int32_t);
  size += (bsz + 1) * sizeof(int);
  size += bsz * slimmed_x_len * sizeof(int32_t);
  return size;
}

template <typename T>
inline void enqueueImpl(const nvinfer1::PluginTensorDesc* input_desc,
                        const nvinfer1::PluginTensorDesc* output_desc,
                        const void* const* inputs,
                        void* const* outputs,
                        void* workspace_ptr,
                        hipStream_t stream,
                        int device_id,
                        T max_value,
                        bool keep_first_token_,
                        bool keep_order_) {
  platform::DeviceContextPool& pool = platform::DeviceContextPool::Instance();
  auto& dev_ctx = *pool.Get(platform::CUDAPlace(device_id));

  // Dims
  auto attn_dims = input_desc[0].dims;
  auto x_dims = input_desc[1].dims;
  auto new_mask_dims = input_desc[3].dims;

  auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
       max_seq_len = attn_dims.d[2];
  auto c = x_dims.d[2];
  auto slimmed_x_len = new_mask_dims.d[2];

  // Inputs
  const T* attn_data = static_cast<const T*>(inputs[0]);
  const T* x_data = static_cast<const T*>(inputs[1]);
  const T* mask_data = static_cast<const T*>(inputs[2]);

  // Outputs
  T* output_data = static_cast<T*>(outputs[0]);
  int32_t* output_indices_data = static_cast<int32_t*>(outputs[1]);

  int total = bsz * nb_head * max_seq_len * max_seq_len;
  int block = operators::ComputeBlockSize(max_seq_len);
  int grid = operators::CeilDivide(total, block);

  // Workspace for intermediate variable
  char* workspace = static_cast<char*>(workspace_ptr);
  T* attn_tmp_data = reinterpret_cast<T*>(workspace);
  size_t offset = total * sizeof(T);
  T* attn_accu_data = reinterpret_cast<T*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(T);
  int32_t* attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(int32_t);
  T* sort_attn_accu_data = reinterpret_cast<T*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(T);
  int32_t* sort_attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);
  offset += bsz * max_seq_len * sizeof(int32_t);
  int* offsets_data = reinterpret_cast<int*>(workspace + offset);
  offset += (bsz + 1) * sizeof(int);
  int32_t* slimmed_sort_attn_accu_indices_data =
      reinterpret_cast<int32_t*>(workspace + offset);

  // 1. Filter attn by mask
  ElementwiseMask<T>
      <<<grid, block, 0, stream>>>(attn_data, mask_data, attn_tmp_data, total);

  total = bsz * max_seq_len;
  block = operators::ComputeBlockSize(max_seq_len);
  grid = operators::CeilDivide(total, block);
  FillZero<T><<<grid, block, 0, stream>>>(attn_accu_data, total);

  // 2. Reduce sum
  total = bsz * nb_head * max_seq_len * max_seq_len;
  int block_tmp = max_seq_len;
  while (block_tmp > 1024)
    block_tmp /= 2;  // if max seq len > 1024, it must be 2^n
  block =
      block_tmp;  // make sure max_seq_len is an integral multiple of block_size
  grid = operators::CeilDivide(total, block);
  ReduceSum2<T><<<grid, block, block * sizeof(T), stream>>>(
      attn_tmp_data, attn_accu_data, bsz, nb_head, max_seq_len);

  // 3. Prepare token indices
  total = bsz * max_seq_len;
  block = operators::ComputeBlockSize(max_seq_len);
  grid = operators::CeilDivide(total, block);

  FillIndex<<<grid, block, 0, stream>>>(
      attn_accu_indices_data, bsz, max_seq_len);

  // 4. Sort token indices by attn
  if (keep_first_token_) {
    MaximumFirst<T>
        <<<bsz, 1, 0, stream>>>(attn_accu_data, bsz, max_seq_len, max_value);
  }
  size_t temp_storage_bytes = -1;
  int num_items = bsz * max_seq_len;
  int num_segments = bsz;
  FillOffsets<<<bsz + 1, 1, 0, stream>>>(offsets_data, bsz, max_seq_len);
  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr,
      temp_storage_bytes,
      attn_accu_data,
      sort_attn_accu_data,
      attn_accu_indices_data,
      sort_attn_accu_indices_data,
      num_items,
      num_segments,
      offsets_data,
      offsets_data + 1,
      0,
      sizeof(T) * 8,
      stream));
  int64_t temp_size = temp_storage_bytes;
  phi::DenseTensor temp_storage;
  temp_storage.Resize({temp_size});
  auto* temp_storage_data = dev_ctx.Alloc<uint8_t>(
      &temp_storage, temp_storage.numel() * sizeof(uint8_t));

  PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      temp_storage_data,
      temp_storage_bytes,
      attn_accu_data,
      sort_attn_accu_data,
      attn_accu_indices_data,
      sort_attn_accu_indices_data,
      num_items,
      num_segments,
      offsets_data,
      offsets_data + 1,
      0,
      sizeof(T) * 8,
      stream));
  // 5. Slice
  total = bsz * slimmed_x_len;
  block = operators::ComputeBlockSize(slimmed_x_len);
  grid = operators::CeilDivide(total, block);

  Slice<int32_t>
      <<<grid, block, 0, stream>>>(sort_attn_accu_indices_data,
                                   slimmed_sort_attn_accu_indices_data,
                                   bsz,
                                   max_seq_len,
                                   slimmed_x_len);

  if (keep_order_) {
    // 6. reorder
    num_items = bsz * slimmed_x_len;
    FillOffsets<<<bsz + 1, 1, 0, stream>>>(offsets_data, bsz, slimmed_x_len);
    temp_storage_bytes = -1;
    PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
        nullptr,
        temp_storage_bytes,
        slimmed_sort_attn_accu_indices_data,
        output_indices_data,
        num_items,
        num_segments,
        offsets_data,
        offsets_data + 1,
        0,
        sizeof(int32_t) * 8,
        stream));

    temp_size = temp_storage_bytes;
    temp_storage.Resize({temp_size});
    temp_storage_data = dev_ctx.Alloc<uint8_t>(
        &temp_storage, temp_storage.numel() * sizeof(uint8_t));
    PADDLE_ENFORCE_GPU_SUCCESS(hipcub::DeviceSegmentedRadixSort::SortKeys(
        temp_storage_data,
        temp_storage_bytes,
        slimmed_sort_attn_accu_indices_data,
        output_indices_data,
        num_items,
        num_segments,
        offsets_data,
        offsets_data + 1,
        0,
        sizeof(int32_t) * 8,
        stream));

    TakeAlongAxis<T><<<grid, block, 0, stream>>>(x_data,
                                                 output_data,
                                                 output_indices_data,
                                                 bsz,
                                                 max_seq_len,
                                                 slimmed_x_len,
                                                 c);
  } else {
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(output_indices_data,
                                          slimmed_sort_attn_accu_indices_data,
                                          bsz * slimmed_x_len * sizeof(int32_t),
                                          hipMemcpyDeviceToDevice));
    TakeAlongAxis<T>
        <<<grid, block, 0, stream>>>(x_data,
                                     output_data,
                                     slimmed_sort_attn_accu_indices_data,
                                     bsz,
                                     max_seq_len,
                                     slimmed_x_len,
                                     c);
  }
}

int FusedTokenPrunePluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc,
    const void* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  if (flag_varseqlen_) {
    if (!(input_desc[0].type == nvinfer1::DataType::kHALF &&
          input_desc[1].type == nvinfer1::DataType::kHALF)) {
      PADDLE_THROW(
          platform::errors::InvalidArgument("Token_prune'type must half"));
    }
    float scale =
        static_cast<float>(input_desc[3].dims.d[2]) / input_desc[6].dims.d[1];
    const int32_t* inputs5 =
        static_cast<const int32_t*>(inputs[5]);             // pre pos id
    int32_t* outputs3 = static_cast<int32_t*>(outputs[3]);  // new pos id
    half* outputs0 = static_cast<half*>(outputs[0]);

    const int32_t B = input_desc[1].dims.d[0];  // batchs
    const int32_t max_sequnce_length =
        input_desc[1].dims.d[1];                     // max sequnce length
    const int32_t length = input_desc[1].dims.d[2];  // vector length
    const half* scores = static_cast<const half*>(inputs[0]);  // reduce sum
    const half* tokens = static_cast<const half*>(inputs[1]);
    const int32_t scores_size = B * max_sequnce_length;
    int32_t padding_token_length;
    if (max_sequnce_length <= 128) {
      padding_token_length = 128;
    } else if (max_sequnce_length <= 256) {
      padding_token_length = 256;
    } else if (max_sequnce_length <= 384) {
      padding_token_length = 384;
    } else {
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Token_prune'token_length must <= 384"));
    }

    // 1. Compute the token length after pruning.
    compute_token_length<<<1, B, 0, stream>>>(
        inputs5, pruned_token_lengths_, scale);

    fill_index_padding_score<<<B, padding_token_length, 0, stream>>>(
        token_index_, scores, scores_size, padding_scores_);

    // Determine temporary device storage requirements
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  pruned_token_lengths_,
                                  outputs3,
                                  B + 1);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  pruned_token_lengths_,
                                  outputs3,
                                  B + 1);

    if (padding_token_length == 128) {
      general_topk_pair_sort<half, 32, 4>
          <<<B, 32, 0, stream>>>(padding_scores_, token_index_);  // 128
    } else if (padding_token_length == 256) {
      general_topk_pair_sort<half, 64, 4>
          <<<B, 64, 0, stream>>>(padding_scores_, token_index_);  // 256
    } else {
      general_topk_pair_sort<half, 96, 4>
          <<<B, 96, 0, stream>>>(padding_scores_, token_index_);  // 384
    }

    int32_t num_threads;
    if (length < 1024) {
      num_threads = length;
    } else {
      if (length % 512 == 0) {
        num_threads = 512;
      } else if (length % 256 == 0) {
        num_threads = 256;
      } else if (length % 128 == 0) {
        num_threads = 128;
      } else if (length % 64 == 0) {
        num_threads = 64;
      } else if (length % 32 == 0) {
        num_threads = 32;
      } else if (length % 16 == 0) {
        num_threads = 16;
      } else if (length % 8 == 0) {
        num_threads = 8;
      } else if (length % 4 == 0) {
        num_threads = 4;
      } else if (length % 2 == 0) {
        num_threads = 2;
      } else {
        num_threads = 1;
      }
    }
    const dim3 num_blocks(
        B,
        max_sequnce_length,
        length / num_threads);  //  batchs, max_sequnce_length, vector_ength/***
    varlen_prune_token<<<num_blocks, num_threads, 0, stream>>>(
        tokens, outputs3, token_index_, outputs0);
  } else {
    auto input_type = input_desc[0].type;
    auto attn_dims = input_desc[0].dims;
    auto bsz = attn_dims.d[0], nb_head = attn_dims.d[1],
         max_seq_len = attn_dims.d[2];
    int device_id;
    hipGetDevice(&device_id);

    if (input_type == nvinfer1::DataType::kFLOAT) {
      VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp32";

      float max = std::numeric_limits<float>::max();

      enqueueImpl<float>(input_desc,
                         output_desc,
                         inputs,
                         outputs,
                         workspace,
                         stream,
                         device_id,
                         max,
                         keep_first_token_,
                         keep_order_);

    } else if (input_type == nvinfer1::DataType::kHALF) {
      VLOG(1) << "TRT Plugin DataType selected. FusedTokenPrune-->fp16";

      half max = 65504.0;
      enqueueImpl<half>(input_desc,
                        output_desc,
                        inputs,
                        outputs,
                        workspace,
                        stream,
                        device_id,
                        max,
                        keep_first_token_,
                        keep_order_);
    } else {
      PADDLE_THROW(
          platform::errors::Fatal("The FusedTokenPrune TRT Plugin's input type "
                                  "should be float or half."));
    }
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
