#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
// SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION &
// AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <iostream>
#include <vector>

#include "NvInfer.h"
#include "paddle/fluid/inference/tensorrt/plugin/common/common.cuh"
#include "paddle/fluid/inference/tensorrt/plugin/prompt_tuning_emb_layernorm_varseqlen_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

template <typename T, unsigned TPB>
__global__ void prompt_tuning_embKernel(int32_t B,
                                        int32_t ld,
                                        int32_t const* inputIds0,
                                        int32_t const* inputIds1,
                                        int32_t const* inputIds2,
                                        T const* dense_vector,
                                        float const* beta,
                                        float const* gamma,
                                        T const* mIdsEmbDev0,
                                        T const* mIdsEmbDev1,
                                        T const* mIdsEmbDev2,
                                        int32_t IdsSize0,
                                        int32_t IdsSize1,
                                        int32_t IdsSize2,
                                        T* output,
                                        int32_t* new_pos_id) {
  hipcub::Sum pairSum;
  int32_t const s = blockIdx.x;
  int32_t const b = blockIdx.y;
  int32_t const sumS = inputIds0[b];
  int32_t const s_b = inputIds0[b + 1] - inputIds0[b];

  int32_t const new_sumS = sumS + b;

  // new pos_id: Add an id to each sentence
  new_pos_id[b] = new_sumS;

  // last id
  if (b == B - 1) {
    new_pos_id[B] = inputIds0[B] + B;
  }

  T const rld = T(1.f) / T(ld);
  int32_t const seqPos = sumS + s;
  int32_t const out_seqPos = new_sumS + s + 1;
  int32_t const new_out_seqPos = new_sumS + s;

  kvp<T> threadData(0, 0);

  int32_t const new_outoffset = new_out_seqPos * ld;
  int32_t const prompt_tuning_offset = new_sumS * ld;
  int32_t const dense_vector_offset = b * ld;

  if (s < s_b) {
    extern __shared__ int32_t word_id[];
    if (threadIdx.x == 0) {
      if (static_cast<int32_t const*>(inputIds1)[seqPos] < 0 ||
          static_cast<int32_t const*>(inputIds1)[seqPos] >= IdsSize1) {
        printf(
            "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
            "table: ID < 0 or ID > max ");
        return;
      } else {
        word_id[0] = static_cast<int32_t const*>(inputIds1)[seqPos];
      }

      if (static_cast<int32_t const*>(inputIds2)[seqPos] < 0 ||
          static_cast<int32_t const*>(inputIds2)[seqPos] >= IdsSize2) {
        printf(
            "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
            "table: ID < 0 or ID > max ");
        return;
      } else {
        word_id[1] = static_cast<int32_t const*>(inputIds2)[seqPos];
      }
    }
    __syncthreads();

    // 2. load pos/tok/word embeddings and add them toghether
    // offset into embeddings is given by wordId * hidden_size
    int32_t const poffset = blockIdx.x * ld;
    int32_t const outoffset = out_seqPos * ld;

    // the output offset is given by b * (S*hidden_size) + s * hidden_size

    for (int32_t it = threadIdx.x; it < ld; it += TPB) {
      T p(mIdsEmbDev0[poffset + it]);  // pos id
      T val = p;
      int32_t const offset0 = word_id[0] * ld;
      val += mIdsEmbDev1[offset0 + it];
      int32_t const offset1 = word_id[1] * ld;
      val += mIdsEmbDev2[offset1 + it];
      output[outoffset + it] = val;
      T const rldval = rld * val;
      threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
    }
    // 3. layer norm on the sum
    layerNorm<T, T, float, TPB>(threadData, ld, outoffset, beta, gamma, output);
  } else if (s == s_b) {
    for (int32_t it = threadIdx.x; it < ld; it += TPB) {
      T val = dense_vector[dense_vector_offset + it];
      output[prompt_tuning_offset + it] = val;
      T const rldval = rld * val;
      threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
      // 3. layer norm on the sum
    }
    layerNorm<T, T, float, TPB>(
        threadData, ld, prompt_tuning_offset, beta, gamma, output);

  } else {
    return;  // This CTA has nothing to do
  }
}

template <typename T>
int32_t prompt_tuning_emb(hipStream_t stream,
                          int32_t ld,
                          int32_t B,
                          int32_t S,
                          int const* inputIds0,
                          int const* inputIds1,
                          int const* inputIds2,
                          T const* dense_vector,
                          int32_t nbLookupTables,
                          float const* beta,
                          float const* gamma,
                          T const* mIdsEmbDev0,
                          T const* mIdsEmbDev1,
                          T const* mIdsEmbDev2,
                          int32_t IdsSize0,
                          int32_t IdsSize1,
                          int32_t IdsSize2,
                          T* output,
                          int32_t* new_pos_id) {
  constexpr int32_t tpb = 256;
  dim3 const grid(S, B, 1);
  dim3 const block(tpb, 1, 1);
  size_t cache_size = sizeof(int32_t) * (nbLookupTables - 1);
  prompt_tuning_embKernel<T, tpb>
      <<<grid, block, cache_size, stream>>>(B,
                                            ld,
                                            inputIds0,
                                            inputIds1,
                                            inputIds2,
                                            dense_vector,
                                            beta,
                                            gamma,
                                            mIdsEmbDev0,
                                            mIdsEmbDev1,
                                            mIdsEmbDev2,
                                            IdsSize0,
                                            IdsSize1,
                                            IdsSize2,
                                            output,
                                            new_pos_id);
  return hipPeekAtLastError();
}

template int32_t prompt_tuning_emb<half>(hipStream_t,
                                         int32_t,
                                         int32_t,
                                         int32_t,
                                         int32_t const*,
                                         int32_t const*,
                                         int32_t const*,
                                         half const*,
                                         int32_t,
                                         float const*,
                                         float const*,
                                         half const*,
                                         half const*,
                                         half const*,
                                         int32_t,
                                         int32_t,
                                         int32_t,
                                         half*,
                                         int32_t*);

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
