// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include "paddle/fluid/inference/tensorrt/plugin/split_op_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {

nvinfer1::Dims SplitPlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims* inputDims,
                                                int nbInputs) {
  assert(nbInputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const& input_dims = inputDims[0];
  nvinfer1::Dims output_dims = input_dims;
  output_dims.d[axis_] = output_length_.at(index);
  return output_dims;
}

int SplitPlugin::initialize() {
  std::vector<int> segment_offsets(1, 0);
  for (int i = 0; i < this->getNbOutputs(); ++i) {
    segment_offsets.push_back(segment_offsets.back() + output_length_[i]);
  }
  segment_offsets_ = segment_offsets;
  d_segment_offsets_ = segment_offsets;
  nvinfer1::Dims dims = this->getInputDims(0);
  nx_ = 1;
  for (int i = dims.nbDims - 1; i > axis_; --i) {
    nx_ *= dims.d[i];
  }
  ny_ = dims.d[axis_];
  nz_ = 1;
  for (int i = axis_ - 1; i >= 0; --i) {
    nz_ *= dims.d[i];
  }
  return 0;
}

int SplitPlugin::enqueue(int batchSize, const void* const* inputs,
                         void** outputs, void* workspace, hipStream_t stream) {
  auto const& input_dims = this->getInputDims(0);
  int input_size = 0;
  int const* d_segment_offsets_ptr =
      thrust::raw_pointer_cast(&d_segment_offsets_[0]);
  float const* idata = reinterpret_cast<float const*>(inputs[0]);
  float** odatas = reinterpret_cast<float**>(outputs);

  // kernel impl here.
  int inputBatchOffset = nx_ * ny_ * nz_;
  for (size_t i = 0; i < this->getNbOutputs(); i++) {
    for (size_t j = 0; j < batchSize; j++) {
      hipMemcpyAsync(
          odatas[i] +
              j * (segment_offsets_[i + 1] - segment_offsets_[i]) * nx_ *
                  sizeof(float),
          inputs[0] +
              (inputBatchOffset * j + segment_offsets_[i] * nx_) *
                  sizeof(float),
          (segment_offsets_[i + 1] - segment_offsets_[i]) * nx_ * sizeof(float),
          hipMemcpyDeviceToDevice, stream);
    }
  }

  return hipGetLastError() != hipSuccess;
}

}  // tensorrt
}  // inference
}  // paddle
