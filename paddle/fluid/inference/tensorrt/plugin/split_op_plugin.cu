#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_fp16.h>
#include <algorithm>
#include "paddle/fluid/inference/tensorrt/plugin/split_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

SplitPlugin* CreateSplitPluginDeserialize(const void* buffer, size_t length) {
  return new SplitPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("split_plugin", CreateSplitPluginDeserialize);

// copied from operators::math::SplitFunctor
template <typename T>
__global__ void SplitKernel(const T* input_data, const int in_row,
                            const int in_col, const int* out_cols,
                            int out_cols_size, T** outputs_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int curr_segment = 0;
  int curr_offset = out_cols[0];
  for (; tid_x < in_col; tid_x += blockDim.x * gridDim.x) {
    int curr_col_offset = out_cols[curr_segment + 1];
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = out_cols[curr_segment + 1];
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;
    T* output_ptr = outputs_data[curr_segment];
    if (output_ptr != nullptr) {
      int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * segment_width + local_col] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__global__ void SplitKernel(const T* input_data, const int in_row,
                            const int in_col, const int fixed_out_col,
                            T** outputs_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  for (; tid_x < in_col; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x / fixed_out_col;
    int in_offset = tid_x - split * fixed_out_col;
    T* output_ptr = outputs_data[split];
    if (output_ptr != nullptr) {
      int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * fixed_out_col + in_offset] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

nvinfer1::Dims SplitPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims* input_dims, int num_inputs) {
  PADDLE_ENFORCE_EQ(num_inputs, 1);
  PADDLE_ENFORCE_LT(index, this->getNbOutputs());

  nvinfer1::Dims output_dims = input_dims[0];
  output_dims.d[axis_] = output_length_.at(index);
  return output_dims;
}

int SplitPlugin::initialize() {
  PADDLE_ENFORCE_LE(axis_, nvinfer1::Dims::MAX_DIMS);
  // notice input dims is [C, H, W]
  nvinfer1::Dims dims = this->getInputDims(0);
  outer_rows_ = 1;
  inner_cols_ = 1;
  for (int i = 0; i < axis_; ++i) {
    outer_rows_ *= dims.d[i];
  }
  for (int i = axis_ + 1; i < dims.nbDims; ++i) {
    inner_cols_ *= dims.d[i];
  }
  same_shape_ = true;
  std::vector<int> segment_offsets(1, 0);
  for (int i = 0; i < this->getNbOutputs(); ++i) {
    if (output_length_[i] != output_length_[0]) {
      same_shape_ = false;
    }
    segment_offsets.push_back(segment_offsets.back() +
                              output_length_[i] * inner_cols_);
  }
  inner_cols_ *= dims.d[axis_];
  d_segment_offsets_ = segment_offsets;
  segment_offsets_ = std::move(segment_offsets);
  d_output_ptrs_.resize(this->getNbOutputs(), nullptr);
  return 0;
}

template <typename T>
inline void Split(hipStream_t stream, const bool same_shape,
                  const int outer_rows, const int inner_cols,
                  const std::vector<int>& segment_offsets,
                  const int* d_segment_offsets, const T* input, T** outputs) {
  const int kThreadsPerBlock = 1024;
  const int kMaxBlocks = 65535;
  int block_cols = kThreadsPerBlock;
  if (inner_cols < kThreadsPerBlock) {  // block_cols is aligned by 32.
    block_cols = ((inner_cols + 31) >> 5) << 5;
  }
  int block_rows = kThreadsPerBlock / block_cols;
  dim3 block_size = dim3(block_cols, block_rows, 1);

  int grid_cols =
      std::min((inner_cols + block_cols - 1) / block_cols, kMaxBlocks);
  int grid_rows =
      std::min(kMaxBlocks / grid_cols, std::max(outer_rows / block_rows, 1));
  dim3 grid_size = dim3(grid_cols, grid_rows, 1);

  if (same_shape) {
    SplitKernel<<<grid_size, block_size, 0, stream>>>(
        input, outer_rows, inner_cols, segment_offsets[1], outputs);
  } else {
    SplitKernel<<<grid_size, block_size, 0, stream>>>(
        input, outer_rows, inner_cols, d_segment_offsets,
        static_cast<int>(segment_offsets.size()), outputs);
  }
}

int SplitPlugin::enqueue(int batchSize, const void* const* inputs,
                         void** outputs, void* workspace, hipStream_t stream) {
  float const* input_ptr = reinterpret_cast<float const*>(inputs[0]);
  if (((batchSize == 1 && axis_ == 0) || axis_ == -1) &&
      this->getNbOutputs() < 10) {
    float** output_ptrs = reinterpret_cast<float**>(outputs);
    int data_type_size = (this->getDataType() == nvinfer1::DataType::kFLOAT)
                             ? sizeof(float)
                             : sizeof(__half);
    for (int i = 0; i < this->getNbOutputs(); ++i) {
      PADDLE_ENFORCE(
          hipMemcpyAsync(
              output_ptrs[i], input_ptr + segment_offsets_[i],
              (segment_offsets_[i + 1] - segment_offsets_[i]) * data_type_size,
              hipMemcpyDeviceToDevice, stream) == hipSuccess);
    }
  } else {
    outer_rows_ *= batchSize;
    const int* d_segment_offsets_ptr =
        thrust::raw_pointer_cast(&d_segment_offsets_[0]);
    float** output_ptrs = thrust::raw_pointer_cast(&d_output_ptrs_[0]);
    PADDLE_ENFORCE(hipMemcpyAsync(output_ptrs, outputs,
                                   this->getNbOutputs() * sizeof(float*),
                                   hipMemcpyHostToDevice,
                                   stream) == hipSuccess);
    if (this->getDataType() == nvinfer1::DataType::kFLOAT) {
      Split(stream, same_shape_, outer_rows_, inner_cols_, segment_offsets_,
            d_segment_offsets_ptr, input_ptr, output_ptrs);
    } else {
      Split(stream, same_shape_, outer_rows_, inner_cols_, segment_offsets_,
            d_segment_offsets_ptr, (__half*)input_ptr,  // NOLINT
            (__half**)output_ptrs);                     // NOLINT
    }
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
