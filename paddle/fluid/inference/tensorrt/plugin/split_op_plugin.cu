#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_fp16.h>
#include <algorithm>
#include "paddle/fluid/inference/tensorrt/plugin/split_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

SplitPlugin* CreateSplitPluginDeserialize(const void* buffer, size_t length) {
  return new SplitPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("split_plugin", CreateSplitPluginDeserialize);

template <typename T>
__device__ int upper_bound(T const* vals, int n, T const& key) {
  int i = 0;
  while (n > 0) {
    int m = n / 2;
    int j = i + m;
    if (!(key < vals[j])) {
      i = j + 1;
      n -= m + 1;
    } else {
      n = m;
    }
  }
  return i;
}

nvinfer1::Dims SplitPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims* input_dims, int num_inputs) {
  PADDLE_ENFORCE_EQ(num_inputs, 1,
                    platform::errors::InvalidArgument(
                        "Invalid number of inputs of split TRT plugin. "
                        "Expected 1, received %d.",
                        num_inputs));
  PADDLE_ENFORCE_LT(
      index, this->getNbOutputs(),
      platform::errors::InvalidArgument(
          "Index of output should be less than the total number of outputs in "
          "split TensorRT plugin. Received index = %d >= total outputs = %d",
          index, this->getNbOutputs()));

  nvinfer1::Dims output_dims = input_dims[0];
  output_dims.d[axis_] = output_length_.at(index);
  return output_dims;
}

int SplitPlugin::initialize() {
  PADDLE_ENFORCE_LE(axis_, nvinfer1::Dims::MAX_DIMS,
                    platform::errors::InvalidArgument(
                        "Axis dimension exceeds max dimension in TensorRT. "
                        "Received axis = %d > MAX_DIMS = %d",
                        axis_, nvinfer1::Dims::MAX_DIMS));
  // notice input dims is [C, H, W]
  nvinfer1::Dims dims = this->getInputDims(0);
  outer_rows_ = 1;
  inner_cols_ = 1;
  for (int i = 0; i < axis_; ++i) {
    outer_rows_ *= dims.d[i];
  }
  for (int i = axis_ + 1; i < dims.nbDims; ++i) {
    inner_cols_ *= dims.d[i];
  }
  same_shape_ = true;
  std::vector<int> segment_offsets(1, 0);
  for (int i = 0; i < this->getNbOutputs(); ++i) {
    if (output_length_[i] != output_length_[0]) {
      same_shape_ = false;
    }
    segment_offsets.push_back(segment_offsets.back() + output_length_[i]);
  }
  axis_shape_ = dims.d[axis_];
  d_segment_offsets_ = segment_offsets;
  segment_offsets_ = std::move(segment_offsets);
  d_output_ptrs_.resize(this->getNbOutputs(), nullptr);
  return 0;
}

// The following part of the code refers to onnx-tensorrt
// https://github.com/onnx/onnx-tensorrt/blob/master/Split.cu
template <typename T>
__global__ void split_kernel(int nsegment,
                             int const* __restrict__ segment_offsets,
                             T const* __restrict__ idata, T* const* odatas,
                             int inner_cols, int axis_shape, int outer_rows) {
  int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  int src_y0 = threadIdx.y + blockIdx.y * blockDim.y;
  int z0 = threadIdx.z + blockIdx.z * blockDim.z;
  for (int z = z0; z < outer_rows; z += blockDim.z * gridDim.z) {
    for (int src_y = src_y0; src_y < axis_shape;
         src_y += blockDim.y * gridDim.y) {
      for (int x = x0; x < inner_cols; x += blockDim.x * gridDim.x) {
        int segment = upper_bound(segment_offsets, nsegment, src_y) - 1;
        int dst_y = src_y - segment_offsets[segment];
        int dst_ny = segment_offsets[segment + 1] - segment_offsets[segment];
        odatas[segment][x + inner_cols * (dst_y + dst_ny * z)] =
            idata[x + inner_cols * (src_y + axis_shape * z)];
      }
    }
  }
}

int SplitPlugin::enqueue(int batchSize, const void* const* inputs,
                         void** outputs, void* workspace, hipStream_t stream) {
  const int* d_segment_offsets_ptr =
      thrust::raw_pointer_cast(&d_segment_offsets_[0]);
  float const* input_ptr = reinterpret_cast<float const*>(inputs[0]);
  float* const* h_odatas = reinterpret_cast<float* const*>(outputs);
  float** output_ptrs = thrust::raw_pointer_cast(&d_output_ptrs_[0]);
  PADDLE_ENFORCE_CUDA_SUCCESS(hipMemcpyAsync(
      output_ptrs, h_odatas, d_output_ptrs_.size() * sizeof(float*),
      hipMemcpyHostToDevice, stream));

  int outer_rows = outer_rows_ * batchSize;

  dim3 block(32, 16);
  dim3 grid(std::min((inner_cols_ - 1) / block.x + 1, 65535u),
            std::min((axis_shape_ - 1) / block.y + 1, 65535u),
            std::min((outer_rows_ - 1) / block.z + 1, 65535u));

  split_kernel<<<grid, block, 0, stream>>>(
      d_segment_offsets_.size(), d_segment_offsets_ptr, input_ptr, output_ptrs,
      inner_cols_, axis_shape_, outer_rows);
  return hipGetLastError() != hipSuccess;
}

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)
int SplitPluginDynamic::initialize() { return 0; }

size_t SplitPluginDynamic::getSerializationSize() const { return 0; }

void SplitPluginDynamic::serialize(void* buffer) const {}

nvinfer1::DimsExprs SplitPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs* inputs, int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) {
  PADDLE_ENFORCE_EQ(nb_inputs, 1,
                    platform::errors::InvalidArgument(
                        "The Split plugin should be only one input."));
  PADDLE_ENFORCE_LT(output_index, output_length_.size(),
                    platform::errors::InvalidArgument(
                        "When GetOutputDimensions, the index(%d) should not "
                        "greater the num(%d) of the outpus.",
                        output_index, output_length_.size()));

  nvinfer1::DimsExprs output_dims = inputs[0];
  output_dims.d[axis_] = expr_builder.constant(output_length_.at(output_index));

  return output_dims;
}

bool SplitPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of split plugin should not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    return (in.type == nvinfer1::DataType::kFLOAT ||
            in.type == nvinfer1::DataType::kHALF) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc& prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType SplitPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types, int nb_inputs) const {
  return input_types[0];
}

int SplitPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* input_desc,
                                const nvinfer1::PluginTensorDesc* output_desc,
                                const void* const* inputs, void* const* outputs,
                                void* workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  int outer_rows = 1;
  int inner_cols = 1;
  // with batch
  for (int i = 0; i < axis_; i++) {
    outer_rows *= input_dims.d[i];
  }

  for (int i = axis_ + 1; i < input_dims.nbDims; i++) {
    inner_cols *= input_dims.d[i];
  }

  std::vector<int> segment_offsets(1, 0);
  for (int i = 0; i < this->getNbOutputs(); i++) {
    segment_offsets.push_back(segment_offsets.back() + output_length_[i]);
  }
  int axis_shape = input_dims.d[axis_];
  thrust::device_vector<int> d_segment_offsets = segment_offsets;
  const int* d_segment_offsets_ptr =
      thrust::raw_pointer_cast(&d_segment_offsets[0]);

  dim3 block(32, 16);
  dim3 grid(std::min((inner_cols - 1) / block.x + 1, 65535u),
            std::min((axis_shape - 1) / block.y + 1, 65535u),
            std::min((outer_rows - 1) / block.z + 1, 65535u));

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    thrust::device_vector<float*> d_output_ptrs;
    d_output_ptrs.resize(this->getNbOutputs(), nullptr);

    const float* input_ptr = static_cast<const float*>(inputs[0]);
    float* const* h_odatas = reinterpret_cast<float* const*>(outputs);
    float** output_ptrs = thrust::raw_pointer_cast(&d_output_ptrs[0]);

    PADDLE_ENFORCE_CUDA_SUCCESS(hipMemcpyAsync(
        output_ptrs, h_odatas, d_output_ptrs.size() * sizeof(float*),
        hipMemcpyHostToDevice, stream));

    split_kernel<<<grid, block, 0, stream>>>(
        d_segment_offsets.size(), d_segment_offsets_ptr, input_ptr, output_ptrs,
        inner_cols, axis_shape, outer_rows);
  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    thrust::device_vector<half*> d_output_ptrs;
    d_output_ptrs.resize(this->getNbOutputs(), nullptr);

    const half* input_ptr = static_cast<const half*>(inputs[0]);
    half* const* h_odatas = reinterpret_cast<half* const*>(outputs);
    half** output_ptrs = thrust::raw_pointer_cast(&d_output_ptrs[0]);

    PADDLE_ENFORCE_CUDA_SUCCESS(hipMemcpyAsync(
        output_ptrs, h_odatas, d_output_ptrs.size() * sizeof(half*),
        hipMemcpyHostToDevice, stream));

    split_kernel<<<grid, block, 0, stream>>>(
        d_segment_offsets.size(), d_segment_offsets_ptr, input_ptr, output_ptrs,
        inner_cols, axis_shape, outer_rows);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
