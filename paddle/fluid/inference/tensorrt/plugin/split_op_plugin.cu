// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/split_op_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

nvinfer1::Dims SplitPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims* input_dims, int num_inputs) {
  PADDLE_ENFORCE_EQ(num_inputs, 1);
  PADDLE_ENFORCE_LT(index, this->getNbOutputs());

  nvinfer1::Dims output_dims = input_dims[0];
  output_dims.d[axis_] = output_length_.at(index);
  return output_dims;
}

int SplitPlugin::initialize() {
  PADDLE_ENFORCE_LE(axis_, nvinfer1::Dims::MAX_DIMS);

  std::vector<int> segment_offsets(1, 0);
  for (int i = 0; i < this->getNbOutputs(); ++i) {
    segment_offsets.push_back(segment_offsets.back() + output_length_[i]);
  }
  segment_offsets_ = segment_offsets;
  nvinfer1::Dims dims = this->getInputDims(0);
  nx_ = 1;
  for (int i = dims.nbDims - 1; i > axis_; --i) {
    nx_ *= dims.d[i];
  }
  ny_ = dims.d[axis_];
  nz_ = 1;
  for (int i = axis_ - 1; i >= 0; --i) {
    nz_ *= dims.d[i];
  }
  return 0;
}

int SplitPlugin::enqueue(int batchSize, const void* const* inputs,
                         void** outputs, void* workspace, hipStream_t stream) {
  auto const& input_dims = this->getInputDims(0);
  int input_size = 0;
  float const* idata = reinterpret_cast<float const*>(inputs[0]);
  float** odatas = reinterpret_cast<float**>(outputs);

  // kernel impl here.
  int inputBatchOffset = nx_ * ny_ * nz_;
  for (size_t i = 0; i < this->getNbOutputs(); i++) {
    for (size_t j = 0; j < batchSize; j++) {
      hipMemcpyAsync(
          odatas[i] +
              j * (segment_offsets_[i + 1] - segment_offsets_[i]) * nx_ *
                  sizeof(float),
          inputs[0] +
              (inputBatchOffset * j + segment_offsets_[i] * nx_) *
                  sizeof(float),
          (segment_offsets_[i + 1] - segment_offsets_[i]) * nx_ * sizeof(float),
          hipMemcpyDeviceToDevice, stream);
    }
  }

  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
