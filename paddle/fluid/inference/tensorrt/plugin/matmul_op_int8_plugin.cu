#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/inference/tensorrt/plugin/matmul_op_int8_plugin.h"

namespace plf = paddle::platform;
namespace dyl = paddle::platform::dynload;
namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
float zero = 0;
void Ltgemm_int8_linear(
    hipblasLtHandle_t ltHandle, const int8_t* A, hipblasLtMatrixLayout_t Adesc,
    int8_t* Atransform, hipblasLtMatrixLayout_t AtransformDesc, bool transA_,
    const int8_t* B, hipblasLtMatrixLayout_t Bdesc, int8_t* Btransform,
    hipblasLtMatrixLayout_t BtransformDesc, bool transB_, int8_t* C,
    hipblasLtMatrixLayout_t Cdesc, int8_t* Ctransform,
    hipblasLtMatrixLayout_t CtransformDesc,
    hipblasLtMatrixTransformDesc_t transformDescT,
    hipblasLtMatrixTransformDesc_t transformDescN,
    hipblasLtMatmulDesc_t matmulDesc, void* alpha_scale, void* alpha_zero,
    void* alpha_one, void* workspace, hipStream_t stream) {
  if (transA_) {
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransform(
        ltHandle, transformDescT, alpha_one, A, Adesc, alpha_zero, nullptr,
        nullptr, Atransform, AtransformDesc, stream));
  } else {
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransform(
        ltHandle, transformDescN, alpha_one, A, Adesc, alpha_zero, nullptr,
        nullptr, Atransform, AtransformDesc, stream));
  }

  if (transB_) {
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransform(
        ltHandle, transformDescN, alpha_one, B, Bdesc, alpha_zero, nullptr,
        nullptr, Btransform, BtransformDesc, stream));
  } else {
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransform(
        ltHandle, transformDescT, alpha_one, B, Bdesc, alpha_zero, nullptr,
        nullptr, Btransform, BtransformDesc, stream));
  }

  PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmul(
      ltHandle, matmulDesc, alpha_scale, Atransform, AtransformDesc, Btransform,
      BtransformDesc, nullptr, Ctransform, CtransformDesc, Ctransform,
      CtransformDesc, nullptr, workspace, 0, stream));

  PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransform(
      ltHandle, transformDescN, alpha_one, Ctransform, CtransformDesc,
      alpha_zero, nullptr, nullptr, C, Cdesc, stream));
}

void Ltgemm_fp32_linear(hipblasLtHandle_t ltHandle, const float* A,
                        hipblasLtMatrixLayout_t Adesc, const float* B,
                        hipblasLtMatrixLayout_t Bdesc, float* C,
                        hipblasLtMatrixLayout_t Cdesc,
                        hipblasLtMatmulDesc_t matmulDesc, void* alpha_scale,
                        void* alpha_zero, void* workspace,
                        hipStream_t stream) {
  PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmul(
      ltHandle, matmulDesc, alpha_scale, A, Adesc, B, Bdesc, alpha_zero, C,
      Cdesc, C, Cdesc, nullptr, workspace, 0, stream));
}

void Ltgemm_fp16_linear(hipblasLtHandle_t ltHandle, const half* A,
                        hipblasLtMatrixLayout_t Adesc, const half* B,
                        hipblasLtMatrixLayout_t Bdesc, half* C,
                        hipblasLtMatrixLayout_t Cdesc,
                        hipblasLtMatmulDesc_t matmulDesc, void* alpha_scale,
                        void* alpha_zero, void* workspace,
                        hipStream_t stream) {
  PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmul(
      ltHandle, matmulDesc, alpha_scale, A, Adesc, B, Bdesc, alpha_zero, C,
      Cdesc, C, Cdesc, nullptr, workspace, 0, stream));
}

nvinfer1::DataType MatmulPlugin::getOutputDataType(
    int index, const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

nvinfer1::Dims MatmulPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims* input_dims, int num_inputs) TRT_NOEXCEPT {
  if (transB_) {
    m_ = dims_x_.d[dims_x_.nbDims - 1];
    k_ = dims_x_.d[dims_x_.nbDims - 2];
  } else {
    m_ = dims_x_.d[dims_x_.nbDims - 2];
    k_ = dims_x_.d[dims_x_.nbDims - 1];
  }
  if (transA_) {
    n_ = dims_y_.d[dims_y_.nbDims - 2];
  } else {
    n_ = dims_y_.d[dims_y_.nbDims - 1];
  }

  batch_ = 1;
  for (int i = 0; i < dims_x_.nbDims - 2; i++) {
    batch_ *= dims_x_.d[i];
  }
  nvinfer1::Dims output_dims;
  output_dims.nbDims = dims_x_.nbDims;
  for (int i = 0; i < output_dims.nbDims - 2; i++) {
    output_dims.d[i] = dims_x_.d[i];
  }
  output_dims.d[output_dims.nbDims - 2] = m_;
  output_dims.d[output_dims.nbDims - 1] = n_;

  return output_dims;
}

bool MatmulPlugin::supportsFormatCombination(
    int32_t pos, nvinfer1::PluginTensorDesc const* inOut, int32_t nbInputs,
    int32_t nbOutputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(nbInputs, 2,
                    platform::errors::InvalidArgument("Must have 2 inputs, "
                                                      "but got %d input(s). ",
                                                      nbInputs));
  PADDLE_ENFORCE_EQ(nbOutputs, getNbOutputs(),
                    platform::errors::InvalidArgument("Must have 1 output, "
                                                      "but got %d output(s). ",
                                                      nbOutputs));
  if (pos == 0) {
    return (inOut[pos].type == nvinfer1::DataType::kHALF ||
            inOut[pos].type == nvinfer1::DataType::kFLOAT ||
            inOut[pos].type == nvinfer1::DataType::kINT8) &&
           inOut[pos].format == nvinfer1::TensorFormat::kLINEAR;
  } else {
    return inOut[pos].type == inOut[0].type &&
           inOut[pos].format == inOut[0].format;
  }
}

void MatmulPlugin::configurePlugin(const nvinfer1::PluginTensorDesc* inputs,
                                   int32_t nbInputs,
                                   const nvinfer1::PluginTensorDesc* out,
                                   int32_t nbOutputs) TRT_NOEXCEPT {
  float inscale_0 = inputs[0].scale;
  float inscale_1 = inputs[1].scale;
  float outscale = out[0].scale;
  type_ = inputs[0].type;
  int64_t stridea = k_ * n_;
  int64_t strideb = k_ * m_;
  int64_t stridec = m_ * n_;

  hipblasOperation_t AopTranspose, BopTranspose;
  if (transA_) {
    AopTranspose = HIPBLAS_OP_T;
  } else {
    AopTranspose = HIPBLAS_OP_N;
  }
  if (transB_) {
    BopTranspose = HIPBLAS_OP_T;
  } else {
    BopTranspose = HIPBLAS_OP_N;
  }

  if (type_ == nvinfer1::DataType::kINT8) {
    hipDataType cudadataTypeIO = HIP_R_8I;
    hipDataType cudaDataTypeS = HIP_R_32F;
#if CUBLAS_VER_MAJOR < 11
    hipDataType cudaComputeType = HIP_R_32I;
#else
    hipblasComputeType_t cudaComputeType = HIPBLAS_COMPUTE_32I;
#endif
    hipblasLtOrder_t COL32 = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    int const ldatransform = 32 * n_;
    int const ldbtransform = 32 * ((m_ + 8 - 1) / 8 * 8);
    int const ldctransform = 32 * n_;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(
        (void**)&Atransform_,
        sizeof(int8_t) * ((k_ + 32 - 1) / 32 * 32) / 32 * ldatransform));
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(
        (void**)&Btransform_,
        sizeof(int8_t) * ((k_ + 32 - 1) / 32 * 32) / 32 * ldbtransform));
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(
        (void**)&Ctransform_,
        sizeof(int8_t) * ((m_ + 32 - 1) / 32 * 32) / 32 * ldctransform));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Adesc_, cudadataTypeIO, AopTranspose == HIPBLAS_OP_N ? n_ : k_,
        AopTranspose == HIPBLAS_OP_N ? k_ : n_,
        AopTranspose == HIPBLAS_OP_N ? n_ : k_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridea),
        sizeof(stridea)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Bdesc_, cudadataTypeIO, BopTranspose == HIPBLAS_OP_N ? k_ : m_,
        BopTranspose == HIPBLAS_OP_N ? m_ : k_,
        BopTranspose == HIPBLAS_OP_N ? k_ : m_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(strideb),
        sizeof(strideb)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixLayoutCreate(&Cdesc_, cudadataTypeIO, n_, m_, n_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridec),
        sizeof(stridec)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &AtransformDesc_, cudadataTypeIO, n_, k_, ldatransform));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        AtransformDesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        AtransformDesc_, HIPBLASLT_MATRIX_LAYOUT_ORDER, &COL32, sizeof(COL32)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &BtransformDesc_, cudadataTypeIO, m_, k_, ldbtransform));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        BtransformDesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        BtransformDesc_, HIPBLASLT_MATRIX_LAYOUT_ORDER, &COL4_4R2_8C,
        sizeof(COL4_4R2_8C)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &CtransformDesc_, cudadataTypeIO, n_, m_, ldctransform));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        CtransformDesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        CtransformDesc_, HIPBLASLT_MATRIX_LAYOUT_ORDER, &COL32, sizeof(COL32)));

    hipblasOperation_t Transpose = HIPBLAS_OP_T;
    hipblasLtPointerMode_t transform_model = HIPBLASLT_POINTER_MODE_DEVICE;
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescCreate(
        &transformDescT_, cudaDataTypeS));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescT_, HIPBLASLT_MATRIX_TRANSFORM_DESC_SCALE_TYPE,
        &cudaDataTypeS, sizeof(cudaDataTypeS)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescT_, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &Transpose,
        sizeof(Transpose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescT_, HIPBLASLT_MATRIX_TRANSFORM_DESC_POINTER_MODE,
        &transform_model, sizeof(transform_model)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescCreate(
        &transformDescN_, cudaDataTypeS));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescN_, HIPBLASLT_MATRIX_TRANSFORM_DESC_SCALE_TYPE,
        &cudaDataTypeS, sizeof(cudaDataTypeS)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescN_, HIPBLASLT_MATRIX_TRANSFORM_DESC_POINTER_MODE,
        &transform_model, sizeof(transform_model)));

    hipblasOperation_t ATranspose = HIPBLAS_OP_N, BTranspose = HIPBLAS_OP_T;
    hipblasLtPointerMode_t matmul_model =
        CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;

#if CUBLAS_VER_MAJOR < 11
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatmulDescCreate(&matmulDesc_, cudaComputeType));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescCreate(
        &matmulDesc_, cudaComputeType, cudaDataTypeS));
#endif

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_TRANSA, &ATranspose,
        sizeof(ATranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_TRANSB, &BTranspose,
        sizeof(BTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &matmul_model,
        sizeof(matmul_model)));

    std::vector<float> alpha_tem(n_, 0);
    for (int i = 0; i < n_; i++) {
      alpha_tem[i] = alpha_ * inscale_0 * inscale_1 / outscale;
    }
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipMalloc((void**)&alpha_scale_, n_ * sizeof(float)));
    hipMemcpyAsync(alpha_scale_, &alpha_tem[0], n_ * sizeof(float),
                    hipMemcpyHostToDevice);
    float zero_tem = zero;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_zero_, sizeof(float)));
    hipMemcpyAsync(alpha_zero_, &zero_tem, sizeof(float),
                    hipMemcpyHostToDevice);
    float one_tem = 1;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_one_, sizeof(float)));
    hipMemcpyAsync(alpha_one_, &one_tem, sizeof(float),
                    hipMemcpyHostToDevice);
  } else if (type_ == nvinfer1::DataType::kHALF) {
    hipDataType cudadataTypeIO = HIP_R_16F;
    hipDataType cudaDataTypeS = HIP_R_16F;
#if CUBLAS_VER_MAJOR < 11
    hipDataType cudaComputeType = HIP_R_16F;
#else
    hipblasComputeType_t cudaComputeType = HIPBLAS_COMPUTE_16F;
#endif
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Adesc_, cudadataTypeIO, AopTranspose == HIPBLAS_OP_N ? n_ : k_,
        AopTranspose == HIPBLAS_OP_N ? k_ : n_,
        AopTranspose == HIPBLAS_OP_N ? n_ : k_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridea),
        sizeof(stridea)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Bdesc_, cudadataTypeIO, BopTranspose == HIPBLAS_OP_N ? k_ : m_,
        BopTranspose == HIPBLAS_OP_N ? m_ : k_,
        BopTranspose == HIPBLAS_OP_N ? k_ : m_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(strideb),
        sizeof(strideb)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixLayoutCreate(&Cdesc_, cudadataTypeIO, n_, m_, n_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridec),
        sizeof(stridec)));

    hipblasLtPointerMode_t matmul_model = HIPBLASLT_POINTER_MODE_DEVICE;

#if CUBLAS_VER_MAJOR < 11
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatmulDescCreate(&matmulDesc_, cudaComputeType));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescCreate(
        &matmulDesc_, cudaComputeType, cudaDataTypeS));
#endif

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_TRANSA, &AopTranspose,
        sizeof(AopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_TRANSB, &BopTranspose,
        sizeof(BopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &matmul_model,
        sizeof(matmul_model)));

    half alpha_tem = static_cast<half>(alpha_);
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_scale_, sizeof(half)));
    hipMemcpyAsync(alpha_scale_, &alpha_tem, sizeof(half),
                    hipMemcpyHostToDevice);
    half zero_tem = static_cast<half>(zero);
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_zero_, sizeof(half)));
    hipMemcpyAsync(alpha_zero_, &zero_tem, sizeof(half),
                    hipMemcpyHostToDevice);
  } else {
    hipDataType cudadataTypeIO = HIP_R_32F;
    hipDataType cudaDataTypeS = HIP_R_32F;
#if CUBLAS_VER_MAJOR < 11
    hipDataType cudaComputeType = HIP_R_32F;
#else
    hipblasComputeType_t cudaComputeType = HIPBLAS_COMPUTE_32F_FAST_16F;
#endif
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Adesc_, cudadataTypeIO, AopTranspose == HIPBLAS_OP_N ? n_ : k_,
        AopTranspose == HIPBLAS_OP_N ? k_ : n_,
        AopTranspose == HIPBLAS_OP_N ? n_ : k_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridea),
        sizeof(stridea)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Bdesc_, cudadataTypeIO, BopTranspose == HIPBLAS_OP_N ? k_ : m_,
        BopTranspose == HIPBLAS_OP_N ? m_ : k_,
        BopTranspose == HIPBLAS_OP_N ? k_ : m_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(strideb),
        sizeof(strideb)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixLayoutCreate(&Cdesc_, cudadataTypeIO, n_, m_, n_));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch_), sizeof(batch_)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc_, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridec),
        sizeof(stridec)));

    hipblasLtPointerMode_t matmul_model = HIPBLASLT_POINTER_MODE_DEVICE;

#if CUBLAS_VER_MAJOR < 11
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatmulDescCreate(&matmulDesc_, cudaComputeType));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescCreate(
        &matmulDesc_, cudaComputeType, cudaDataTypeS));
#endif

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_TRANSA, &AopTranspose,
        sizeof(AopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_TRANSB, &BopTranspose,
        sizeof(BopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc_, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &matmul_model,
        sizeof(matmul_model)));

    float alpha_tem = alpha_;
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipMalloc((void**)&alpha_scale_, sizeof(float)));
    hipMemcpyAsync(alpha_scale_, &alpha_tem, sizeof(float),
                    hipMemcpyHostToDevice);
    float zero_tem = zero;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_zero_, sizeof(float)));
    hipMemcpyAsync(alpha_zero_, &zero_tem, sizeof(float),
                    hipMemcpyHostToDevice);
  }
}

void MatmulPlugin::attachToContext(
    cudnnContext* cudnnContext, cublasContext* cublasContext,
    nvinfer1::IGpuAllocator* gpuAllocator) TRT_NOEXCEPT {
  dyl::hipblasLtCreate(&cublas_);
}

void MatmulPlugin::detachFromContext() TRT_NOEXCEPT {
  dyl::hipblasLtDestroy(cublas_);
}

// When tensorrt engine freed ,there is "double free" ERROR. TODO@Wangzheee
void MatmulPlugin::terminate() TRT_NOEXCEPT {
  /*
   if(alpha_scale_){
     hipFree((void *)alpha_scale_);
     alpha_scale_ = nullptr;
   }
   if(alpha_zero_){
     hipFree((void *)alpha_zero_);
     alpha_zero_ = nullptr;
   }
   if(alpha_one_){
     hipFree((void *)alpha_one_);
     alpha_one_ = nullptr;
   }
   if(Atransform_){
     hipFree((void *)Atransform_);
     Atransform_ = nullptr;
   }
   if(Btransform_){
     hipFree((void *)Btransform_);
     Btransform_ = nullptr;
   }
   if(Ctransform_){
     hipFree((void *)Ctransform_);
     Ctransform_ = nullptr;
   }   */
}

int MatmulPlugin::enqueue(int batchSize, const void* const* inputs,
#if IS_TRT_VERSION_LT(8000)
                          void** outputs, void* workspace,
                          hipStream_t stream) {
#else
                          void* const* outputs, void* workspace,
                          hipStream_t stream) TRT_NOEXCEPT {
#endif
  if (type_ == nvinfer1::DataType::kINT8) {
    const int8_t* B = static_cast<const int8_t*>(inputs[0]);
    const int8_t* A = static_cast<const int8_t*>(inputs[1]);
    int8_t* C = static_cast<int8_t*>(outputs[0]);
    Ltgemm_int8_linear(
        cublas_, A, Adesc_, Atransform_, AtransformDesc_, transA_, B, Bdesc_,
        Btransform_, BtransformDesc_, transB_, C, Cdesc_, Ctransform_,
        CtransformDesc_, transformDescT_, transformDescN_, matmulDesc_,
        alpha_scale_, alpha_zero_, alpha_one_, workspace, stream);
  } else if (type_ == nvinfer1::DataType::kFLOAT) {
    const float* B = static_cast<const float*>(inputs[0]);
    const float* A = static_cast<const float*>(inputs[1]);
    float* C = static_cast<float*>(outputs[0]);
    Ltgemm_fp32_linear(cublas_, A, Adesc_, B, Bdesc_, C, Cdesc_, matmulDesc_,
                       alpha_scale_, alpha_zero_, workspace, stream);
  } else if (type_ == nvinfer1::DataType::kHALF) {
    const half* B = static_cast<const half*>(inputs[0]);
    const half* A = static_cast<const half*>(inputs[1]);
    half* C = static_cast<half*>(outputs[0]);
    Ltgemm_fp16_linear(cublas_, A, Adesc_, B, Bdesc_, C, Cdesc_, matmulDesc_,
                       alpha_scale_, alpha_zero_, workspace, stream);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "VarMessageToVarType:Unsupported type"));
  }
  return hipGetLastError() != hipSuccess;
}

nvinfer1::DataType MatmulPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

nvinfer1::DimsExprs MatmulPluginDynamic::getOutputDimensions(
    int outputIndex, const nvinfer1::DimsExprs* inputs, int nbInputs,
    nvinfer1::IExprBuilder& exprBuilder) TRT_NOEXCEPT {
  nvinfer1::DimsExprs output_dims(inputs[0]);
  if (transB_) {
    output_dims.d[output_dims.nbDims - 2] = inputs[0].d[inputs[0].nbDims - 1];
  } else {
    output_dims.d[output_dims.nbDims - 2] = inputs[0].d[inputs[0].nbDims - 2];
  }
  if (transA_) {
    output_dims.d[output_dims.nbDims - 1] = inputs[1].d[inputs[1].nbDims - 2];
  } else {
    output_dims.d[output_dims.nbDims - 1] = inputs[1].d[inputs[1].nbDims - 1];
  }
  return output_dims;
}

bool MatmulPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* inOut, int nbInputs,
    int nbOutputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(nbInputs, 2,
                    platform::errors::InvalidArgument("Must have 2 inputs, "
                                                      "but got %d input(s). ",
                                                      nbInputs));
  PADDLE_ENFORCE_EQ(nbOutputs, getNbOutputs(),
                    platform::errors::InvalidArgument("Must have 1 output, "
                                                      "but got %d output(s). ",
                                                      nbOutputs));
  if (pos == 0) {
    return (inOut[pos].type == nvinfer1::DataType::kHALF ||
            inOut[pos].type == nvinfer1::DataType::kFLOAT ||
            inOut[pos].type == nvinfer1::DataType::kINT8) &&
           inOut[pos].format == nvinfer1::TensorFormat::kLINEAR;
  } else {
    return inOut[pos].type == inOut[0].type &&
           inOut[pos].format == nvinfer1::TensorFormat::kLINEAR;
  }
}

void MatmulPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* outputs,
    int nbOutputs) TRT_NOEXCEPT {
  float inscale_0 = inputs[0].desc.scale;
  float inscale_1 = inputs[1].desc.scale;
  float outscale = outputs[0].desc.scale;
  type_ = inputs[0].desc.type;
  uint64_t m_max, n_max, k_max;
  if (transB_) {
    m_max = inputs[0].max.d[inputs[0].max.nbDims - 1];
    k_max = inputs[0].max.d[inputs[0].max.nbDims - 2];
  } else {
    m_max = inputs[0].max.d[inputs[0].max.nbDims - 2];
    k_max = inputs[0].max.d[inputs[0].max.nbDims - 1];
  }
  if (transA_) {
    n_max = inputs[1].max.d[inputs[1].max.nbDims - 2];
  } else {
    n_max = inputs[1].max.d[inputs[1].max.nbDims - 1];
  }

  int const ldatransform = 32 * n_max;
  int const ldbtransform = 32 * ((m_max + 8 - 1) / 8 * 8);
  int const ldctransform = 32 * n_max;
  PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(
      (void**)&Atransform_,
      sizeof(int8_t) * ((k_max + 32 - 1) / 32 * 32) / 32 * ldatransform));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(
      (void**)&Btransform_,
      sizeof(int8_t) * ((k_max + 32 - 1) / 32 * 32) / 32 * ldbtransform));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(
      (void**)&Ctransform_,
      sizeof(int8_t) * ((m_max + 32 - 1) / 32 * 32) / 32 * ldctransform));

  if (type_ == nvinfer1::DataType::kINT8) {
    std::vector<float> alpha_tem(n_max, 0);
    for (int i = 0; i < n_max; i++) {
      alpha_tem[i] = alpha_ * inscale_0 * inscale_1 / outscale;
    }
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipMalloc((void**)&alpha_scale_, n_max * sizeof(float)));
    hipMemcpyAsync(alpha_scale_, &alpha_tem[0], n_max * sizeof(float),
                    hipMemcpyHostToDevice);
    float zero_tem = zero;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_zero_, sizeof(float)));
    hipMemcpyAsync(alpha_zero_, &zero_tem, sizeof(float),
                    hipMemcpyHostToDevice);
    float one_tem = 1;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_one_, sizeof(float)));
    hipMemcpyAsync(alpha_one_, &one_tem, sizeof(float),
                    hipMemcpyHostToDevice);
  } else if (type_ == nvinfer1::DataType::kHALF) {
    half alpha_tem = static_cast<half>(alpha_);
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_scale_, sizeof(half)));
    hipMemcpyAsync(alpha_scale_, &alpha_tem, sizeof(half),
                    hipMemcpyHostToDevice);
    half zero_tem = static_cast<half>(zero);
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_zero_, sizeof(half)));
    hipMemcpyAsync(alpha_zero_, &zero_tem, sizeof(half),
                    hipMemcpyHostToDevice);
  } else {
    float alpha_tem = alpha_;
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipMalloc((void**)&alpha_scale_, sizeof(float)));
    hipMemcpyAsync(alpha_scale_, &alpha_tem, sizeof(float),
                    hipMemcpyHostToDevice);
    float zero_tem = zero;
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc((void**)&alpha_zero_, sizeof(float)));
    hipMemcpyAsync(alpha_zero_, &zero_tem, sizeof(float),
                    hipMemcpyHostToDevice);
  }
}

void MatmulPluginDynamic::attachToContext(
    cudnnContext* cudnnContext, cublasContext* cublasContext,
    nvinfer1::IGpuAllocator* gpuAllocator) TRT_NOEXCEPT {
  dyl::hipblasLtCreate(&cublas_);
}

void MatmulPluginDynamic::detachFromContext() TRT_NOEXCEPT {
  dyl::hipblasLtDestroy(cublas_);
}

// When tensorrt engine freed ,there is "double free" ERROR. TODO@Wangzheee
void MatmulPluginDynamic::terminate() TRT_NOEXCEPT {
  /*if(alpha_scale_){
    hipFree((void *)alpha_scale_);
    alpha_scale_ = nullptr;
  }
  if(alpha_zero_){
    hipFree((void *)alpha_zero_);
    alpha_zero_ = nullptr;
  }
  if(alpha_one_){
    hipFree((void *)alpha_one_);
    alpha_one_ = nullptr;
  }
  if(Atransform_){
    hipFree((void *)Atransform_);
    Atransform_ = nullptr;
  }
  if(Btransform_){
    hipFree((void *)Btransform_);
    Btransform_ = nullptr;
  }
  if(Ctransform_){
    hipFree((void *)Ctransform_);
    Ctransform_ = nullptr;
  } */
}

int MatmulPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
                                 const nvinfer1::PluginTensorDesc* outputDesc,
                                 const void* const* inputs,
                                 void* const* outputs, void* workspace,
                                 hipStream_t stream) TRT_NOEXCEPT {
  const auto Input0Desc = inputDesc[0];
  const auto Input1Desc = inputDesc[1];
  uint64_t m, n, k;
  if (transB_) {
    m = Input0Desc.dims.d[Input0Desc.dims.nbDims - 1];
    k = Input0Desc.dims.d[Input0Desc.dims.nbDims - 2];
  } else {
    m = Input0Desc.dims.d[Input0Desc.dims.nbDims - 2];
    k = Input0Desc.dims.d[Input0Desc.dims.nbDims - 1];
  }
  if (transA_) {
    n = Input1Desc.dims.d[Input1Desc.dims.nbDims - 2];
  } else {
    n = Input1Desc.dims.d[Input1Desc.dims.nbDims - 1];
  }

  int batch = 1;
  for (int i = 0; i < Input0Desc.dims.nbDims - 2; i++) {
    batch *= Input0Desc.dims.d[i];
  }
  int const ldatransform = 32 * n;
  int const ldbtransform = 32 * ((m + 8 - 1) / 8 * 8);
  int const ldctransform = 32 * n;

  int64_t stridea = k * n;
  int64_t strideb = k * m;
  int64_t stridec = m * n;

  hipblasOperation_t AopTranspose, BopTranspose;
  if (transA_) {
    AopTranspose = HIPBLAS_OP_T;
  } else {
    AopTranspose = HIPBLAS_OP_N;
  }
  if (transB_) {
    BopTranspose = HIPBLAS_OP_T;
  } else {
    BopTranspose = HIPBLAS_OP_N;
  }

  hipblasLtMatrixLayout_t Adesc{nullptr}, Bdesc{nullptr}, Cdesc{nullptr};
  hipblasLtMatmulDesc_t matmulDesc{nullptr};
  hipblasLtMatrixLayout_t AtransformDesc{nullptr}, BtransformDesc{nullptr},
      CtransformDesc{nullptr};
  int8_t *Atransform{nullptr}, *Btransform{nullptr}, *Ctransform{nullptr};
  hipblasLtMatrixTransformDesc_t transformDescT{nullptr},
      transformDescN{nullptr};
  if (type_ == nvinfer1::DataType::kINT8) {
    hipDataType cudadataTypeIO = HIP_R_8I;
    hipDataType cudaDataTypeS = HIP_R_32F;
#if CUBLAS_VER_MAJOR < 11
    hipDataType cudaComputeType = HIP_R_32I;
#else
    hipblasComputeType_t cudaComputeType = HIPBLAS_COMPUTE_32I;
#endif
    hipblasLtOrder_t COL32 = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Adesc, cudadataTypeIO, AopTranspose == HIPBLAS_OP_N ? n : k,
        AopTranspose == HIPBLAS_OP_N ? k : n,
        AopTranspose == HIPBLAS_OP_N ? n : k));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridea),
        sizeof(stridea)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Bdesc, cudadataTypeIO, BopTranspose == HIPBLAS_OP_N ? k : m,
        BopTranspose == HIPBLAS_OP_N ? m : k,
        BopTranspose == HIPBLAS_OP_N ? k : m));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(strideb),
        sizeof(strideb)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixLayoutCreate(&Cdesc, cudadataTypeIO, n, m, n));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridec),
        sizeof(stridec)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &AtransformDesc, cudadataTypeIO, n, k, ldatransform));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &COL32, sizeof(COL32)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &BtransformDesc, cudadataTypeIO, m, k, ldbtransform));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &COL4_4R2_8C,
        sizeof(COL4_4R2_8C)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &CtransformDesc, cudadataTypeIO, n, m, ldctransform));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &COL32, sizeof(COL32)));

    hipblasOperation_t Transpose = HIPBLAS_OP_T;
    hipblasLtPointerMode_t transform_model = HIPBLASLT_POINTER_MODE_DEVICE;
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixTransformDescCreate(&transformDescT, cudaDataTypeS));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescT, HIPBLASLT_MATRIX_TRANSFORM_DESC_SCALE_TYPE,
        &cudaDataTypeS, sizeof(cudaDataTypeS)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescT, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &Transpose,
        sizeof(Transpose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescT, HIPBLASLT_MATRIX_TRANSFORM_DESC_POINTER_MODE,
        &transform_model, sizeof(transform_model)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixTransformDescCreate(&transformDescN, cudaDataTypeS));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescN, HIPBLASLT_MATRIX_TRANSFORM_DESC_SCALE_TYPE,
        &cudaDataTypeS, sizeof(cudaDataTypeS)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixTransformDescSetAttribute(
        transformDescN, HIPBLASLT_MATRIX_TRANSFORM_DESC_POINTER_MODE,
        &transform_model, sizeof(transform_model)));

    hipblasOperation_t ATranspose = HIPBLAS_OP_N, BTranspose = HIPBLAS_OP_T;
    hipblasLtPointerMode_t matmul_model =
        CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;

#if CUBLAS_VER_MAJOR < 11
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatmulDescCreate(&matmulDesc, cudaComputeType));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescCreate(
        &matmulDesc, cudaComputeType, cudaDataTypeS));
#endif

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &ATranspose,
        sizeof(ATranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &BTranspose,
        sizeof(BTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &matmul_model,
        sizeof(matmul_model)));

    const int8_t* B = static_cast<const int8_t*>(inputs[0]);
    const int8_t* A = static_cast<const int8_t*>(inputs[1]);
    int8_t* C = static_cast<int8_t*>(outputs[0]);
    Ltgemm_int8_linear(cublas_, A, Adesc, Atransform_, AtransformDesc, transA_,
                       B, Bdesc, Btransform_, BtransformDesc, transB_, C, Cdesc,
                       Ctransform_, CtransformDesc, transformDescT,
                       transformDescN, matmulDesc, alpha_scale_, alpha_zero_,
                       alpha_one_, workspace, stream);
  } else if (type_ == nvinfer1::DataType::kHALF) {
    hipDataType cudadataTypeIO = HIP_R_16F;
    hipDataType cudaDataTypeS = HIP_R_16F;
#if CUBLAS_VER_MAJOR < 11
    hipDataType cudaComputeType = HIP_R_16F;
#else
    hipblasComputeType_t cudaComputeType = HIPBLAS_COMPUTE_16F;
#endif
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Adesc, cudadataTypeIO, AopTranspose == HIPBLAS_OP_N ? n : k,
        AopTranspose == HIPBLAS_OP_N ? k : n,
        AopTranspose == HIPBLAS_OP_N ? n : k));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridea),
        sizeof(stridea)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Bdesc, cudadataTypeIO, BopTranspose == HIPBLAS_OP_N ? k : m,
        BopTranspose == HIPBLAS_OP_N ? m : k,
        BopTranspose == HIPBLAS_OP_N ? k : m));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(strideb),
        sizeof(strideb)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixLayoutCreate(&Cdesc, cudadataTypeIO, n, m, n));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridec),
        sizeof(stridec)));

    hipblasLtPointerMode_t matmul_model = HIPBLASLT_POINTER_MODE_DEVICE;

#if CUBLAS_VER_MAJOR < 11
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatmulDescCreate(&matmulDesc, cudaComputeType));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescCreate(
        &matmulDesc, cudaComputeType, cudaDataTypeS));
#endif

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &AopTranspose,
        sizeof(AopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &BopTranspose,
        sizeof(BopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &matmul_model,
        sizeof(matmul_model)));

    const half* B = static_cast<const half*>(inputs[0]);
    const half* A = static_cast<const half*>(inputs[1]);
    half* C = static_cast<half*>(outputs[0]);
    Ltgemm_fp16_linear(cublas_, A, Adesc, B, Bdesc, C, Cdesc, matmulDesc,
                       alpha_scale_, alpha_zero_, workspace, stream);
  } else {
    hipDataType cudadataTypeIO = HIP_R_32F;
    hipDataType cudaDataTypeS = HIP_R_32F;
#if CUBLAS_VER_MAJOR < 11
    hipDataType cudaComputeType = HIP_R_32F;
#else
    hipblasComputeType_t cudaComputeType = HIPBLAS_COMPUTE_32F_FAST_16F;
#endif
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Adesc, cudadataTypeIO, AopTranspose == HIPBLAS_OP_N ? n : k,
        AopTranspose == HIPBLAS_OP_N ? k : n,
        AopTranspose == HIPBLAS_OP_N ? n : k));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridea),
        sizeof(stridea)));

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutCreate(
        &Bdesc, cudadataTypeIO, BopTranspose == HIPBLAS_OP_N ? k : m,
        BopTranspose == HIPBLAS_OP_N ? m : k,
        BopTranspose == HIPBLAS_OP_N ? k : m));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(strideb),
        sizeof(strideb)));

    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatrixLayoutCreate(&Cdesc, cudadataTypeIO, n, m, n));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_TYPE, &cudadataTypeIO,
        sizeof(cudadataTypeIO)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &(batch), sizeof(batch)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatrixLayoutSetAttribute(
        Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &(stridec),
        sizeof(stridec)));

    hipblasLtPointerMode_t matmul_model = HIPBLASLT_POINTER_MODE_DEVICE;

#if CUBLAS_VER_MAJOR < 11
    PADDLE_ENFORCE_GPU_SUCCESS(
        dyl::hipblasLtMatmulDescCreate(&matmulDesc, cudaComputeType));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescCreate(
        &matmulDesc, cudaComputeType, cudaDataTypeS));
#endif

    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &AopTranspose,
        sizeof(AopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &BopTranspose,
        sizeof(BopTranspose)));
    PADDLE_ENFORCE_GPU_SUCCESS(dyl::hipblasLtMatmulDescSetAttribute(
        matmulDesc, HIPBLASLT_MATMUL_DESC_POINTER_MODE, &matmul_model,
        sizeof(matmul_model)));

    const float* B = static_cast<const float*>(inputs[0]);
    const float* A = static_cast<const float*>(inputs[1]);
    float* C = static_cast<float*>(outputs[0]);
    Ltgemm_fp32_linear(cublas_, A, Adesc, B, Bdesc, C, Cdesc, matmulDesc,
                       alpha_scale_, alpha_zero_, workspace, stream);
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
