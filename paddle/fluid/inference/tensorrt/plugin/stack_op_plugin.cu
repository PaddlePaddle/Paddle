#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include <vector>
#include "paddle/fluid/inference/tensorrt/plugin/stack_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

#if IS_TRT_VERSION_GE(6000)
StackPluginDynamic::StackPluginDynamic(int axis, int num_stack)
    : axis_(axis), num_stack_(num_stack) {}

StackPluginDynamic::StackPluginDynamic(void const* serial_data,
                                       size_t serial_length) {
  DeserializeValue(&serial_data, &serial_length, &axis_);
  DeserializeValue(&serial_data, &serial_length, &num_stack_);
}

StackPluginDynamic::~StackPluginDynamic() {}

nvinfer1::IPluginV2DynamicExt* StackPluginDynamic::clone() const {
  return new StackPluginDynamic(axis_, num_stack_);
}

const char* StackPluginDynamic::getPluginType() const { return "stack_plugin"; }

int StackPluginDynamic::getNbOutputs() const { return 1; }

int StackPluginDynamic::initialize() { return 0; }

size_t StackPluginDynamic::getSerializationSize() const {
  size_t serialize_size = 0;
  serialize_size += SerializedSize(axis_);
  serialize_size += SerializedSize(num_stack_);
  return serialize_size;
}

void StackPluginDynamic::serialize(void* buffer) const {
  SerializeValue(&buffer, axis_);
  SerializeValue(&buffer, num_stack_);
}

nvinfer1::DimsExprs StackPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs* inputs, int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) {
  nvinfer1::DimsExprs output(inputs[0]);
  output.nbDims = inputs[0].nbDims + 1;

  for (int i = inputs[0].nbDims; i > axis_; --i) {
    output.d[i] = inputs[0].d[i - 1];
  }
  output.d[axis_] = expr_builder.constant(nb_inputs);
  return output;
}

void StackPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs) {}

size_t StackPluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const {
  return num_stack_ * sizeof(uintptr_t);
}

void StackPluginDynamic::destroy() { delete this; }

void StackPluginDynamic::terminate() {}

bool StackPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of stack plugin should not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    return (in.type == nvinfer1::DataType::kFLOAT ||
            in.type == nvinfer1::DataType::kHALF) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc& prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType StackPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The index should be equal to 0"));
  return input_types[0];
}

template <typename T>
__global__ void StackKernel(const T* const* input, T* output, int num_stack,
                            int base_unit) {
  int stack_id = blockIdx.x;
  int lead_id = blockIdx.y;

  for (int i = threadIdx.x; i < base_unit; i += blockDim.x) {
    output[lead_id * num_stack * base_unit + stack_id * base_unit + i] =
        input[stack_id][lead_id * base_unit + i];
  }
}

int StackPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* input_desc,
                                const nvinfer1::PluginTensorDesc* output_desc,
                                const void* const* inputs, void* const* outputs,
                                void* workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;  // (batch, seq, seq)
  auto out_dims = output_desc[0].dims;   // (batch, num_head, seq, seq)
  auto out_num_dims = out_dims.nbDims;

  int base_unit = 1;
  for (int i = axis_ + 1; i < out_num_dims; ++i) {
    PADDLE_ENFORCE_GT(out_dims.d[i], 0,
                      platform::errors::InvalidArgument(
                          "Input dimensions should be greater than 0"));
    base_unit *= out_dims.d[i];
  }

  int lead_unit = 1;
  for (int i = 0; i < axis_; ++i) {
    PADDLE_ENFORCE_GT(out_dims.d[i], 0,
                      platform::errors::InvalidArgument(
                          "Input dimensions should be greater than 0"));
    lead_unit *= out_dims.d[i];
  }

  PADDLE_ENFORCE_EQ(
      out_dims.d[axis_], num_stack_,
      platform::errors::InvalidArgument("number of stack axis should be same"));

  hipMemcpyAsync(workspace, reinterpret_cast<const void* const>(inputs),
                  sizeof(void*) * out_dims.d[axis_], hipMemcpyHostToDevice,
                  stream);

  const int num_stacks = out_dims.d[axis_];
  dim3 num_blocks(num_stacks, lead_unit);
  const int num_threads = 256;
  auto infer_type = input_desc[0].type;

  if (infer_type == nvinfer1::DataType::kFLOAT) {
    float* output = static_cast<float*>(outputs[0]);
    StackKernel<float><<<num_blocks, num_threads, 0, stream>>>(
        reinterpret_cast<const float* const*>(workspace), output, num_stacks,
        base_unit);
  } else if (infer_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    __half* output = static_cast<__half*>(outputs[0]);
    StackKernel<__half><<<num_blocks, num_threads, 0, stream>>>(
        reinterpret_cast<const __half* const*>(workspace), output, num_stacks,
        base_unit);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The Stack TRT Plugin's input type only "
                                "support float or half currently."));
  }
  return hipGetLastError() != hipSuccess;
}

StackPluginDynamicCreator::StackPluginDynamicCreator() {}

const char* StackPluginDynamicCreator::getPluginName() const {
  return "stack_plugin";
}

const char* StackPluginDynamicCreator::getPluginVersion() const { return "1"; }

const nvinfer1::PluginFieldCollection*
StackPluginDynamicCreator::getFieldNames() {
  return &field_collection_;
}

nvinfer1::IPluginV2* StackPluginDynamicCreator::createPlugin(
    const char* name, const nvinfer1::PluginFieldCollection* fc) {
  int axis = -1;
  int num_stack = -1;

  for (int i = 0; i < fc->nbFields; ++i) {
    const std::string name(fc->fields[i].name);
    if (name == "axis") {
      axis = static_cast<const int*>(fc->fields[i].data)[0];
    } else if (name == "num_stack") {
      num_stack = static_cast<const int*>(fc->fields[i].data)[0];
    } else {
      PADDLE_THROW(platform::errors::Fatal("Meet an unknown plugin field '" +
                                           name +
                                           "' when creating stack op plugin."));
    }
  }
  return new StackPluginDynamic(axis, num_stack);
}

nvinfer1::IPluginV2* StackPluginDynamicCreator::deserializePlugin(
    const char* name, const void* serial_data, size_t serial_length) {
  auto plugin = new StackPluginDynamic(serial_data, serial_length);
  return plugin;
}

void StackPluginDynamicCreator::setPluginNamespace(const char* lib_namespace) {
  plugin_namespace_ = lib_namespace;
}

const char* StackPluginDynamicCreator::getPluginNamespace() const {
  return plugin_namespace_.c_str();
}

#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
