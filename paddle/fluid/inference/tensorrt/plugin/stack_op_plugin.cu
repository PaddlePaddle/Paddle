#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include <vector>
#include "paddle/fluid/inference/tensorrt/plugin/stack_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)
size_t StackPluginDynamic::getSerializationSize() const { return 0; }

void StackPluginDynamic::serialize(void* buffer) const {}

nvinfer1::DimsExprs StackPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs* inputs, int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) {
  nvinfer1::DimsExprs output(inputs[0]);
  output.nbDims = inputs[0].nbDims + 1;

  for (int i = inputs[0].nbDims; i > axis_; --i) {
    output.d[i] = inputs[0].d[i - 1];
  }
  output.d[axis_] = expr_builder.constant(nb_inputs);
  return output;
}

bool StackPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of stack plugin should not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc& in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    return (in.type == nvinfer1::DataType::kFLOAT ||
            in.type == nvinfer1::DataType::kHALF) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc& prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType StackPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The index should be equal to 0"));
  return input_types[0];
}

template <typename T>
__global__ void StackKernel(const T* const* input, T* output, int num_stack,
                            int base_unit) {
  int stack_id = blockIdx.x;
  int lead_id = blockIdx.y;

  for (int i = threadIdx.x; i < base_unit; i += blockDim.x) {
    output[lead_id * num_stack * base_unit + stack_id * base_unit + i] =
        input[stack_id][lead_id * base_unit + i];
  }
}

int StackPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* input_desc,
                                const nvinfer1::PluginTensorDesc* output_desc,
                                const void* const* inputs, void* const* outputs,
                                void* workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;  // (batch, seq, seq)
  auto out_dims = output_desc[0].dims;   // (batch, num_head, seq, seq)
  auto out_num_dims = out_dims.nbDims;

  int base_unit = 1;
  for (int i = axis_ + 1; i < out_num_dims; ++i) {
    PADDLE_ENFORCE_GT(out_dims.d[i], 0,
                      platform::errors::InvalidArgument(
                          "Input dimensions should be greater than 0"));
    base_unit *= out_dims.d[i];
  }

  int lead_unit = 1;
  for (int i = 0; i < axis_; ++i) {
    PADDLE_ENFORCE_GT(out_dims.d[i], 0,
                      platform::errors::InvalidArgument(
                          "Input dimensions should be greater than 0"));
    lead_unit *= out_dims.d[i];
  }

  hipMemcpyAsync(reinterpret_cast<void*>(in_ptr_gpu_),
                  reinterpret_cast<const void* const>(inputs),
                  sizeof(void*) * out_dims.d[axis_], hipMemcpyHostToDevice,
                  stream);

  int num_stacks = out_dims.d[axis_];
  dim3 num_blocks(num_stacks, lead_unit);
  int num_threads = 256;
  auto infer_type = input_desc[0].type;

  if (infer_type == nvinfer1::DataType::kFLOAT) {
    float* output = static_cast<float*>(outputs[0]);
    StackKernel<float><<<num_blocks, num_threads, 0, stream>>>(
        reinterpret_cast<const float* const*>(in_ptr_gpu_), output, num_stacks,
        base_unit);
  } else if (infer_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    __half* output = static_cast<__half*>(outputs[0]);
    StackKernel<__half><<<num_blocks, num_threads, 0, stream>>>(
        reinterpret_cast<const __half* const*>(in_ptr_gpu_), output, num_stacks,
        base_unit);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The Stack TRT Plugin's input type only "
                                "support float or half currently."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
