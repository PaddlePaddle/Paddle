#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include "paddle/fluid/inference/tensorrt/plugin/hard_swish_op_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

nvinfer1::Dims HardSwishPlugin::getOutputDimensions(
    int index, const nvinfer1::Dims *in_dims, int nb_inputs) TRT_NOEXCEPT {
  assert(nb_inputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const &input_dims = in_dims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

template <typename T>
__device__ T kMax(T a, T b) {
  return a > b ? a : b;
}

template <typename T>
__device__ T kMin(T a, T b) {
  return a < b ? a : b;
}

template <typename T, unsigned TPB>
__global__ void hard_swish_kernel(float threshold, float scale, float offset,
                                  int n, const T *input, T *output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;
  if (idx < n) {
    const T in = input[idx];
    output[idx] = in / scale * kMin<T>(kMax<T>(in + offset, 0), threshold);
  }
}

int HardSwishPlugin::enqueue(int batch_size, const void *const *inputs,
#if IS_TRT_VERSION_LT(8000)
                             void **outputs, void *, hipStream_t stream) {
#else
                             void *const *outputs, void *,
                             hipStream_t stream) TRT_NOEXCEPT {
#endif
  const auto &input_dims = this->getInputDims(0);
  int num = batch_size;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  float threshold = threshold_;
  float scale = scale_;
  float offset = offset_;

  const int block_size = 256;
  const int grid_size = (num + block_size - 1) / block_size;

  const float *input = static_cast<const float *>(inputs[0]);
  float *output = static_cast<float *>(outputs[0]);
  hard_swish_kernel<float, block_size><<<grid_size, block_size, 0, stream>>>(
      threshold, scale, offset, num, input, output);

  return hipGetLastError() != hipSuccess;
}

#if IS_TRT_VERSION_GE(6000)

nvinfer1::DimsExprs HardSwishPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  return inputs[0];
}

int HardSwishPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  int num = 1;
  for (int i = 0; i < input_dims.nbDims; i++) {
    num *= input_dims.d[i];
  }
  float threshold = threshold_;
  float scale = scale_;
  float offset = offset_;
  const int block_size = 256;
  const int grid_size = (num + block_size - 1) / block_size;
  const float *input = static_cast<const float *>(inputs[0]);
  float *output = static_cast<float *>(outputs[0]);
  hard_swish_kernel<float, block_size><<<grid_size, block_size, 0, stream>>>(
      threshold, scale, offset, num, input, output);

  return hipGetLastError() != hipSuccess;
}

nvinfer1::DataType HardSwishPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(index, 0,
                    platform::errors::InvalidArgument(
                        "The Elementwise Plugin only has one input, so the "
                        "index value should be 0, but get %d.",
                        index));
  return input_types[0];
}

bool HardSwishPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}
#endif
}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
