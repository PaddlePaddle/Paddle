#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
// Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "paddle/fluid/inference/tensorrt/plugin/split_gelu_op_plugin.h"
#include <vector>

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

template <int32_t HHS, int32_t TPB>
__global__ void splitGeLUKernelHalf(half2 const *input,
                                    half2 *output,
                                    float const fDiv,
                                    float const fAdd,
                                    float const fMul) {
  int32_t indexInput = blockIdx.x * HHS * 2 + threadIdx.x;
  int32_t indexOutput = blockIdx.x * HHS + threadIdx.x;
  const int loop = HHS / TPB;
  const half2 hadd(1, 1);
  const half2 hmul(0.5, 0.5);
#pragma unroll
  for (int32_t i = 0; i < loop; ++i) {
    half2 hvalueL = input[indexInput];
    auto hvalueR = input[indexInput + HHS];
    auto tmp = __half22float2(hvalueR);
    tmp.x /= 1.4140625f;
    tmp.y /= 1.4140625f;
    tmp.x = erff(tmp.x);
    tmp.y = erff(tmp.y);
    half2 htmp = __float22half2_rn(tmp);
    htmp = __hadd2(htmp, hadd);
    htmp = __hmul2(htmp, hvalueR);
    htmp = __hmul2(htmp, hmul);
    htmp = __hmul2(htmp, hvalueL);
    output[indexOutput] = htmp;
    indexInput += TPB;
    indexOutput += TPB;
  }
}

template <typename T>
int32_t launchSplitGeLUKernel(hipStream_t stream,
                              int32_t gridSize,
                              int32_t nHalfHiddenSize,
                              T const *input,
                              T *output,
                              float const fDiv,
                              float const fAdd,
                              float const fMul) {
  constexpr int32_t TPB = 256;  // thread per block
  switch (nHalfHiddenSize) {
    case 1280:
      (splitGeLUKernelHalf<640, 64>)<<<gridSize, 64, 0, stream>>>(
          input, output, fDiv, fAdd, fMul);
      break;
    case 2560:
      (splitGeLUKernelHalf<1280, 128>)<<<gridSize, 128, 0, stream>>>(
          input, output, fDiv, fAdd, fMul);
      break;
    case 5120:
      (splitGeLUKernelHalf<2560, 256>)<<<gridSize, 256, 0, stream>>>(
          input, output, fDiv, fAdd, fMul);
      break;
    default:
      PADDLE_THROW(
          platform::errors::Fatal("The function launchSplitGeLUKernel of "
                                  "SplitGeluPluginDynamic TRT Plugin "
                                  "encounter error"));
      break;
  }
  return 0;
}

void SplitGeluPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc *in,
    int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc *out,
    int nbOutputs) TRT_NOEXCEPT {}
bool SplitGeluPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
      return in.type == nvinfer1::DataType::kHALF &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    } else {
      return in.type == nvinfer1::DataType::kFLOAT &&
             in.format == nvinfer1::TensorFormat::kLINEAR;
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType SplitGeluPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

nvinfer1::DimsExprs SplitGeluPluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  nvinfer1::DimsExprs output = inputs[0];
  output.d[2] = expr_builder.operation(nvinfer1::DimensionOperation::kFLOOR_DIV,
                                       *inputs[0].d[2],
                                       *expr_builder.constant(2));
  return output;
}
int SplitGeluPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  const auto &input_dims = input_desc[0].dims;
  auto input_type = input_desc[0].type;
  const int32_t gridSize = input_desc[0].dims.d[0] * input_desc[0].dims.d[1];
  const int32_t nHalfHiddenSize = input_desc[0].dims.d[2] / 2;  // HHS
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(3) << "TRT Plugin DataType selected. SplitGelu-->fp32";
    /*    launchSplitGeLUKernel(stream,
                              gridSize,
                              nHalfHiddenSize,
                              static_cast<float const *>(inputs[0]),
                              static_cast<float *>(outputs[0]),
                              1.4140625f,
                              1,
                              0.5f);*/
  } else if (input_type == nvinfer1::DataType::kHALF) {
    VLOG(3) << "TRT Plugin DataType selected. SplitGelu-->fp16";
    launchSplitGeLUKernel(stream,
                          gridSize,
                          nHalfHiddenSize,
                          static_cast<half2 const *>(inputs[0]),
                          static_cast<half2 *>(outputs[0]),
                          1.4140625f,
                          1,
                          0.5f);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "The SplitGelu TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
