#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES.
All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/inference/tensorrt/plugin/ms_deform_attn_op_plugin.h"
#include <hipcub/hipcub.hpp>
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/kernels/funcs/math_function.h"
//#include <hip/hip_fp16.h>

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {



const int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N, const int num_threads) {
  return (N + num_threads - 1) / num_threads;
}

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__device__ __forceinline__ T warp_shfl_xor(T value, int laneMask, int width,
                                           unsigned int mask = 0xffffffff) {
#if CUDA_VERSION >= 9000
  return __shfl_xor_sync(mask, value, laneMask, width);
#else
  return __shfl_xor(value, laneMask, width);
#endif
}

template <typename T, int width>
__device__ __forceinline__ void warp_reduce(T *sum) {
  for (int offset = width / 2; offset > 0; offset /= 2) {
    T b = warp_shfl_xor(*sum, offset, width);
  }
}

template <typename scalar_t>
__device__ scalar_t ms_deform_attn_im2col_bilinear(
    const scalar_t *&bottom_data, const int &height, const int &width,
    const int &nheads, const int &channels, const scalar_t &h,
    const scalar_t &w, const int &m, const int &c, const int h_stride,
    const int w_stride, const int base_ptr) {
  const int h_low = floorf(h);
  const int w_low = floorf(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const scalar_t lh = h - static_cast<scalar_t>(h_low);
  const scalar_t lw = w - static_cast<scalar_t>(w_low);
  const scalar_t hh = static_cast<scalar_t>(1) - lh,
                 hw = static_cast<scalar_t>(1) - lw;

  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;

  scalar_t v1 = static_cast<scalar_t>(0);
  if (h_low >= 0 && w_low >= 0) {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  scalar_t v2 = static_cast<scalar_t>(0);
  if (h_low >= 0 && w_high <= width - 1) {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  scalar_t v3 = static_cast<scalar_t>(0);
  if (h_high <= height - 1 && w_low >= 0) {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  scalar_t v4 = static_cast<scalar_t>(0);
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ void ms_deform_attn_col2im_bilinear(
    const scalar_t *&bottom_data, const int &height, const int &width,
    const int &nheads, const int &channels, const scalar_t &h,
    const scalar_t &w, const int &m, const int &c, const scalar_t &top_grad,
    const scalar_t &attn_weight, scalar_t *&grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight, const int w_stride,
    const int h_stride, const int base_ptr) {
  const int h_low = floorf(h);
  const int w_low = floorf(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  const scalar_t top_grad_value = top_grad * attn_weight;
  scalar_t grad_h_weight = 0, grad_w_weight = 0;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0) {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
    grad_h_weight -= hw * v1;
    grad_w_weight -= hh * v1;
    atomicAdd(grad_value + ptr1, w1 * top_grad_value);
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
    grad_h_weight -= lw * v2;
    grad_w_weight += hh * v2;
    atomicAdd(grad_value + ptr2, w2 * top_grad_value);
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
    grad_h_weight += hw * v3;
    grad_w_weight -= lh * v3;
    atomicAdd(grad_value + ptr3, w3 * top_grad_value);
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
    grad_h_weight += lw * v4;
    grad_w_weight += lh * v4;
    atomicAdd(grad_value + ptr4, w4 * top_grad_value);
  }

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  *grad_attn_weight = top_grad * val;
  *grad_sampling_loc = width * grad_w_weight * top_grad_value;
  *(grad_sampling_loc + 1) = height * grad_h_weight * top_grad_value;
}

template <typename scalar_t>
__device__ void ms_deform_attn_col2im_bilinear_gm(
    const scalar_t *&bottom_data, const int &height, const int &width,
    const int &nheads, const int &channels, const scalar_t &h,
    const scalar_t &w, const int &m, const int &c, const scalar_t &top_grad,
    const scalar_t &attn_weight, scalar_t *&grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  const int h_low = floorf(h);
  const int w_low = floorf(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const int w_stride = nheads * channels;
  const int h_stride = width * w_stride;
  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int base_ptr = m * channels + c;

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  const scalar_t top_grad_value = top_grad * attn_weight;
  scalar_t grad_h_weight = 0, grad_w_weight = 0;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0) {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
    grad_h_weight -= hw * v1;
    grad_w_weight -= hh * v1;
    atomicAdd(grad_value + ptr1, w1 * top_grad_value);
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
    grad_h_weight -= lw * v2;
    grad_w_weight += hh * v2;
    atomicAdd(grad_value + ptr2, w2 * top_grad_value);
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
    grad_h_weight += hw * v3;
    grad_w_weight -= lh * v3;
    atomicAdd(grad_value + ptr3, w3 * top_grad_value);
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
    grad_h_weight += lw * v4;
    grad_w_weight += lh * v4;
    atomicAdd(grad_value + ptr4, w4 * top_grad_value);
  }

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  atomicAdd(grad_attn_weight, top_grad * val);
  atomicAdd(grad_sampling_loc, width * grad_w_weight * top_grad_value);
  atomicAdd(grad_sampling_loc + 1, height * grad_h_weight * top_grad_value);
}

template <typename scalar_t>
__global__ void ms_deformable_im2col_gpu_kernel(
    const int n, const scalar_t *data_value, const int *data_spatial_shapes,
    const int *data_level_start_index, const scalar_t *data_sampling_loc,
    const scalar_t *data_attn_weight, const int batch_size,
    const int spatial_size, const int num_heads, const int channels,
    const int num_levels, const int num_query, const int num_point,
    scalar_t *data_col) {
  const int w_stride = num_heads * channels;
  CUDA_KERNEL_LOOP(index, n) {
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const int base_ptr = m_col * channels + c_col;

    scalar_t *data_col_ptr = data_col + index;
    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;
    scalar_t col = static_cast<scalar_t>(0);

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int h_stride = spatial_w * w_stride;
      const scalar_t *data_value_ptr =
          data_value +
          (data_value_ptr_init_offset + level_start_id * qid_stride);
      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * static_cast<scalar_t>(spatial_h) -
                              static_cast<scalar_t>(0.5);
        const scalar_t w_im = loc_w * static_cast<scalar_t>(spatial_w) -
                              static_cast<scalar_t>(0.5);

        if (h_im > static_cast<scalar_t>(-1) &&
            w_im > static_cast<scalar_t>(-1) &&
            h_im < static_cast<scalar_t>(spatial_h) &&
            w_im < static_cast<scalar_t>(spatial_w)) {
          col += ms_deform_attn_im2col_bilinear(
                     data_value_ptr, spatial_h, spatial_w, num_heads, channels,
                     h_im, w_im, m_col, c_col, h_stride, w_stride, base_ptr) *
                 weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
    }
    *data_col_ptr = col;
  }
}

template <typename scalar_t, unsigned int blockSize>
__global__ void ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1(
    const int n, const scalar_t *grad_col, const scalar_t *data_value,
    const int *data_spatial_shapes, const int *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int w_stride = num_heads * channels;
  CUDA_KERNEL_LOOP(index, n) {
    __shared__ scalar_t cache_grad_sampling_loc[blockSize * 2];
    __shared__ scalar_t cache_grad_attn_weight[blockSize];
    unsigned int tid = threadIdx.x;
    int _temp = index;
    const int c_col = _temp & (blockSize - 1);
    _temp /= blockSize;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const int base_ptr = m_col * channels + c_col;

    const scalar_t top_grad = grad_col[index];

    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int grad_weight_stride = 1;
    const int grad_loc_stride = 2;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int h_stride = spatial_w * w_stride;
      const int value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          ms_deform_attn_col2im_bilinear(
              data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im,
              w_im, m_col, c_col, top_grad, weight, grad_value_ptr,
              cache_grad_sampling_loc + (threadIdx.x << 1),
              cache_grad_attn_weight + threadIdx.x, h_stride, w_stride,
              base_ptr);
        }

        __syncthreads();

        scalar_t _grad_w = cache_grad_sampling_loc[threadIdx.x],
                 _grad_h = cache_grad_sampling_loc[threadIdx.x + 1],
                 _grad_a = cache_grad_attn_weight[threadIdx.x];

        warp_reduce<scalar_t, blockSize>(&_grad_w);
        warp_reduce<scalar_t, blockSize>(&_grad_h);
        warp_reduce<scalar_t, blockSize>(&_grad_a);

        if (tid == 0) {
          *grad_sampling_loc = _grad_w;
          *(grad_sampling_loc + 1) = _grad_h;
          *grad_attn_weight = _grad_a;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t, unsigned int blockSize>
__global__ void ms_deformable_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2(
    const int n, const scalar_t *grad_col, const scalar_t *data_value,
    const int *data_spatial_shapes, const int *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int w_stride = num_heads * channels;
  CUDA_KERNEL_LOOP(index, n) {
    __shared__ scalar_t cache_grad_sampling_loc[blockSize * 2];
    __shared__ scalar_t cache_grad_attn_weight[blockSize];
    unsigned int tid = threadIdx.x;
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const int base_ptr = m_col * channels + c_col;

    const scalar_t top_grad = grad_col[index];

    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int grad_weight_stride = 1;
    const int grad_loc_stride = 2;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int h_stride = spatial_w * w_stride;
      const int value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          ms_deform_attn_col2im_bilinear(
              data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im,
              w_im, m_col, c_col, top_grad, weight, grad_value_ptr,
              cache_grad_sampling_loc + (threadIdx.x << 1),
              cache_grad_attn_weight + threadIdx.x, h_stride, w_stride,
              base_ptr);
        }

        __syncthreads();

        for (unsigned int s = blockSize / 2; s > 0; s >>= 1) {
          if (tid < s) {
            const unsigned int xid1 = tid << 1;
            const unsigned int xid2 = (tid + s) << 1;
            cache_grad_attn_weight[tid] += cache_grad_attn_weight[tid + s];
            cache_grad_sampling_loc[xid1] += cache_grad_sampling_loc[xid2];
            cache_grad_sampling_loc[xid1 + 1] +=
                cache_grad_sampling_loc[xid2 + 1];
          }
          __syncthreads();
        }

        if (tid == 0) {
          *grad_sampling_loc = cache_grad_sampling_loc[0];
          *(grad_sampling_loc + 1) = cache_grad_sampling_loc[1];
          *grad_attn_weight = cache_grad_attn_weight[0];
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void ms_deformable_col2im_gpu_kernel_shm_reduce_v1(
    const int n, const scalar_t *grad_col, const scalar_t *data_value,
    const int *data_spatial_shapes, const int *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int w_stride = num_heads * channels;
  CUDA_KERNEL_LOOP(index, n) {
    extern __shared__ int _s[];
    scalar_t *cache_grad_sampling_loc = reinterpret_cast<scalar_t *>(_s);
    scalar_t *cache_grad_attn_weight = cache_grad_sampling_loc + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const int base_ptr = m_col * channels + c_col;

    const scalar_t top_grad = grad_col[index];

    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int grad_weight_stride = 1;
    const int grad_loc_stride = 2;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int h_stride = spatial_w * w_stride;
      const int value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          ms_deform_attn_col2im_bilinear(
              data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im,
              w_im, m_col, c_col, top_grad, weight, grad_value_ptr,
              cache_grad_sampling_loc + (threadIdx.x << 1),
              cache_grad_attn_weight + threadIdx.x, h_stride, w_stride,
              base_ptr);
        }

        __syncthreads();
        if (tid == 0) {
          scalar_t _grad_w = cache_grad_sampling_loc[0],
                   _grad_h = cache_grad_sampling_loc[1],
                   _grad_a = cache_grad_attn_weight[0];
          int sid = 2;
          for (unsigned int tid = 1; tid < blockDim.x; ++tid) {
            _grad_w += cache_grad_sampling_loc[sid];
            _grad_h += cache_grad_sampling_loc[sid + 1];
            _grad_a += cache_grad_attn_weight[tid];
            sid += 2;
          }

          *grad_sampling_loc = _grad_w;
          *(grad_sampling_loc + 1) = _grad_h;
          *grad_attn_weight = _grad_a;
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void ms_deformable_col2im_gpu_kernel_shm_reduce_v2(
    const int n, const scalar_t *grad_col, const scalar_t *data_value,
    const int *data_spatial_shapes, const int *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int w_stride = num_heads * channels;
  CUDA_KERNEL_LOOP(index, n) {
    extern __shared__ int _s[];
    scalar_t *cache_grad_sampling_loc = reinterpret_cast<scalar_t *>(_s);
    scalar_t *cache_grad_attn_weight = cache_grad_sampling_loc + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const int base_ptr = m_col * channels + c_col;

    const scalar_t top_grad = grad_col[index];

    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int grad_weight_stride = 1;
    const int grad_loc_stride = 2;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int h_stride = spatial_w * w_stride;
      const int value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          ms_deform_attn_col2im_bilinear(
              data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im,
              w_im, m_col, c_col, top_grad, weight, grad_value_ptr,
              cache_grad_sampling_loc + (threadIdx.x << 1),
              cache_grad_attn_weight + threadIdx.x, h_stride, w_stride,
              base_ptr);
        }

        __syncthreads();

        for (unsigned int s = blockDim.x / 2, spre = blockDim.x; s > 0;
             s >>= 1, spre >>= 1) {
          if (tid < s) {
            const unsigned int xid1 = tid << 1;
            const unsigned int xid2 = (tid + s) << 1;
            cache_grad_attn_weight[tid] += cache_grad_attn_weight[tid + s];
            cache_grad_sampling_loc[xid1] += cache_grad_sampling_loc[xid2];
            cache_grad_sampling_loc[xid1 + 1] +=
                cache_grad_sampling_loc[xid2 + 1];
            if (tid + (s << 1) < spre) {
              cache_grad_attn_weight[tid] +=
                  cache_grad_attn_weight[tid + (s << 1)];
              cache_grad_sampling_loc[xid1] +=
                  cache_grad_sampling_loc[xid2 + (s << 1)];
              cache_grad_sampling_loc[xid1 + 1] +=
                  cache_grad_sampling_loc[xid2 + 1 + (s << 1)];
            }
          }
          __syncthreads();
        }

        if (tid == 0) {
          *grad_sampling_loc = cache_grad_sampling_loc[0];
          *(grad_sampling_loc + 1) = cache_grad_sampling_loc[1];
          *grad_attn_weight = cache_grad_attn_weight[0];
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void ms_deformable_col2im_gpu_kernel_shm_reduce_v2_multi_blocks(
    const int n, const scalar_t *grad_col, const scalar_t *data_value,
    const int *data_spatial_shapes, const int *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int w_stride = num_heads * channels;
  CUDA_KERNEL_LOOP(index, n) {
    extern __shared__ int _s[];
    scalar_t *cache_grad_sampling_loc = reinterpret_cast<scalar_t *>(_s);
    scalar_t *cache_grad_attn_weight = cache_grad_sampling_loc + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const int base_ptr = m_col * channels + c_col;

    const scalar_t top_grad = grad_col[index];

    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int grad_weight_stride = 1;
    const int grad_loc_stride = 2;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int h_stride = spatial_w * w_stride;
      const int value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          ms_deform_attn_col2im_bilinear(
              data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im,
              w_im, m_col, c_col, top_grad, weight, grad_value_ptr,
              cache_grad_sampling_loc + (threadIdx.x << 1),
              cache_grad_attn_weight + threadIdx.x, h_stride, w_stride,
              base_ptr);
        }

        __syncthreads();

        for (unsigned int s = blockDim.x / 2, spre = blockDim.x; s > 0;
             s >>= 1, spre >>= 1) {
          if (tid < s) {
            const unsigned int xid1 = tid << 1;
            const unsigned int xid2 = (tid + s) << 1;
            cache_grad_attn_weight[tid] += cache_grad_attn_weight[tid + s];
            cache_grad_sampling_loc[xid1] += cache_grad_sampling_loc[xid2];
            cache_grad_sampling_loc[xid1 + 1] +=
                cache_grad_sampling_loc[xid2 + 1];
            if (tid + (s << 1) < spre) {
              cache_grad_attn_weight[tid] +=
                  cache_grad_attn_weight[tid + (s << 1)];
              cache_grad_sampling_loc[xid1] +=
                  cache_grad_sampling_loc[xid2 + (s << 1)];
              cache_grad_sampling_loc[xid1 + 1] +=
                  cache_grad_sampling_loc[xid2 + 1 + (s << 1)];
            }
          }
          __syncthreads();
        }

        if (tid == 0) {
          atomicAdd(grad_sampling_loc, cache_grad_sampling_loc[0]);
          atomicAdd(grad_sampling_loc + 1, cache_grad_sampling_loc[1]);
          atomicAdd(grad_attn_weight, cache_grad_attn_weight[0]);
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void ms_deformable_col2im_gpu_kernel_gm(
    const int n, const scalar_t *grad_col, const scalar_t *data_value,
    const int *data_spatial_shapes, 
    const int *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int batch_size, const int spatial_size, const int num_heads,
    const int channels, const int num_levels, const int num_query,
    const int num_point, scalar_t *grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    _temp /= num_query;
    const int b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int grad_weight_stride = 1;
    const int grad_loc_stride = 2;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          ms_deform_attn_col2im_bilinear_gm(
              data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im,
              w_im, m_col, c_col, top_grad, weight, grad_value_ptr,
              grad_sampling_loc, grad_attn_weight);
        }
        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}


nvinfer1::DimsExprs MsDeformAttnPluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputDims,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
//     value_shape,
//     sampling_locations_shape,
//     attention_weights_shape,
//     spatial_shapes_shape,
//     level_start_index_shape) {
//   {{value_shape[0], sampling_locations_shape[1], value_shape[2] * value_shape[3]}};

  nvinfer1::DimsExprs output;
  output.nbDims = 3;
  output.d[0] = inputDims[0].d[0];
  output.d[1] = inputDims[1].d[1];
  output.d[2] = expr_builder.operation(nvinfer1::DimensionOperation::kPROD,
                                    *inputDims[0].d[2],
                                    *inputDims[0].d[3]);
  return output;
}

bool MsDeformAttnPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    // nvinfer1::DataType::kHALF will result diff
      return (in.type == nvinfer1::DataType::kFLOAT || in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
  } else if (pos == 3 || pos == 4) {
      return (in.type == nvinfer1::DataType::kINT32) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
  } 
  const nvinfer1::PluginTensorDesc &prev = in_out[0];
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType MsDeformAttnPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

int MsDeformAttnPluginDynamic::initialize() TRT_NOEXCEPT { return 0; }

int MsDeformAttnPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {

auto value = inputs[0];
auto value_dims = input_desc[0].dims;
auto sampling_locations = inputs[1];
auto sampling_locations_dims = input_desc[1].dims;
auto attention_weights = inputs[2];
auto attention_weights_dims = input_desc[2].dims;
auto spatial_shapes = inputs[3];
auto spatial_shapes_dims = input_desc[3].dims;
auto level_start_index = inputs[4];

const int batch = value_dims.d[0];
const int spatial_size = value_dims.d[1];
const int num_heads = value_dims.d[2];
const int channels = value_dims.d[3];

const int num_levels = spatial_shapes_dims.d[0];

const int num_query = sampling_locations_dims.d[1];
const int num_point = sampling_locations_dims.d[4];

const int im2col_step_new = std::min(batch, im2col_step_);
assert(batch % im2col_step_new == 0);
//   auto output = paddle::full({batch, num_query, num_heads * channels}, 0,
//                              value.type(), paddle::GPUPlace());
hipMemsetAsync(outputs[0], 0, batch * num_query * num_heads * channels * sizeof(float), stream);

  auto per_value_size = spatial_size * num_heads * channels;
  auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
  auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
  auto per_output_size = num_query * num_heads * channels;

  auto input_type = input_desc[0].type;

  for (int n = 0; n < batch / im2col_step_new; ++n) {
    const int num_kernels = im2col_step_new * per_output_size;
    const int num_actual_kernels = im2col_step_new * per_output_size;
    const int num_threads = CUDA_NUM_THREADS;
if(input_type == nvinfer1::DataType::kFLOAT){
    ms_deformable_im2col_gpu_kernel<float>
        <<<GET_BLOCKS(num_actual_kernels, num_threads), 
           num_threads, 
           0,
           stream>>>(
            num_kernels,
            (float*)(value) + n * im2col_step_new * per_value_size,
            (int*)(spatial_shapes), 
            (int*)(level_start_index),
            (float*)(sampling_locations) + n * im2col_step_new * per_sample_loc_size,
            (float*)(attention_weights) + n * im2col_step_new * per_attn_weight_size,
            im2col_step_new, 
            spatial_size, 
            num_heads, 
            channels, 
            num_levels,
            num_query, 
            num_point,
            (float*)(outputs[0]) + n * im2col_step_new * per_output_size);
} else if (input_type == nvinfer1::DataType::kHALF) {
      ms_deformable_im2col_gpu_kernel<half>
        <<<GET_BLOCKS(num_actual_kernels, num_threads), 
           num_threads, 
           0,
           stream>>>(
            num_kernels,
            (half*)(value) + n * im2col_step_new * per_value_size,
            (int*)(spatial_shapes), 
            (int*)(level_start_index),
            (half*)(sampling_locations) + n * im2col_step_new * per_sample_loc_size,
            (half*)(attention_weights) + n * im2col_step_new * per_attn_weight_size,
            im2col_step_new, 
            spatial_size, 
            num_heads, 
            channels, 
            num_levels,
            num_query, 
            num_point,
            (half*)(outputs[0]) + n * im2col_step_new * per_output_size);
}          
  }


      return hipGetLastError() != hipSuccess;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
