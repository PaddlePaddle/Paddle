#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/inference/tensorrt/plugin/qkv_to_context_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_utils.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"
#include "paddle/fluid/operators/math/blas.h"
#include "paddle/fluid/platform/device_context.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

template <typename T>
__global__ void transpose(T *src, T *dst, const int batch_size,
                          const int seq_len, const int head_num,
                          const int size_per_head) {
  int batch_id = blockIdx.x / (head_num * seq_len);
  int seq_id = blockIdx.x % seq_len;
  int head_id = (blockIdx.x % (head_num * seq_len)) / seq_len;
  dst[batch_id * (head_num * seq_len * size_per_head) +
      seq_id * head_num * size_per_head + head_id * size_per_head +
      threadIdx.x] = src[blockIdx.x * size_per_head + threadIdx.x];
}

template <typename T>
__global__ void TransposeQkvKernel(const int H, const T *input, T *output) {
  // Input: BxSx3xNxH
  // Bias: 3xSxB
  // Output: 3xBxNxSxH
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;

  const int N = blockDim.y;
  const int S = gridDim.x;
  const int B = gridDim.y;

  const int NH = N * H;
  const int NHS = NH * S;
  const int in_offset = n * H + m * NH + s * 3 * NH + b * NHS * 3;
  const int out_offset = s * H + n * S * H + b * NHS + m * NHS * B;

  const int i = threadIdx.x;
  output[out_offset + i] = input[in_offset + i];
}

inline void TransposeQKV(const int batch, const int seq_len,
                         const int head_size, const int head_num,
                         const float *input, float *output,
                         hipStream_t stream) {
  int scratch_size = batch * head_num * seq_len * seq_len;
  const dim3 grid(seq_len, batch, 3);
  if (head_size % 4 == 0 && scratch_size % 4 == 0) {
    const int h = head_size / 4;
    const float4 *input4 = reinterpret_cast<const float4 *>(input);
    float4 *output4 = reinterpret_cast<float4 *>(output);
    const dim3 block(h, head_num, 1);
    // limit h * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(h * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024 * 4));
    TransposeQkvKernel<float4><<<grid, block, 0, stream>>>(h, input4, output4);
  } else if (head_size % 2 == 0 && scratch_size % 2 == 0) {
    const int h = head_size / 2;
    const float2 *input2 = reinterpret_cast<const float2 *>(input);
    float2 *output2 = reinterpret_cast<float2 *>(output);
    const dim3 block(h, head_num, 1);
    // limit h * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(h * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024 * 2));
    TransposeQkvKernel<float2><<<grid, block, 0, stream>>>(h, input2, output2);
  } else {
    const dim3 block(head_size, head_num, 1);
    // limit head_size * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(head_size * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024));
    TransposeQkvKernel<float><<<grid, block, 0, stream>>>(head_size, input,
                                                          output);
  }
}

#ifdef SUPPORTS_CUDA_FP16
inline void TransposeQKV(const int batch, const int seq_len,
                         const int head_size, const int head_num,
                         const half *input, half *output, hipStream_t stream) {
  int scratch_size = batch * head_num * seq_len * seq_len;
  const dim3 grid(seq_len, batch, 3);
  if (head_size % 8 == 0 && scratch_size % 8 == 0) {
    int h = head_size / 8;
    const int4 *input4 = reinterpret_cast<const int4 *>(input);
    int4 *output4 = reinterpret_cast<int4 *>(output);
    dim3 block(h, head_num, 1);
    // limit h * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(h * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024 * 8));
    TransposeQkvKernel<int4><<<grid, block, 0, stream>>>(h, input4, output4);
  } else if (head_size % 2 == 0 && scratch_size % 2 == 0) {
    const int h = head_size / 2;
    const half2 *input2 = reinterpret_cast<const half2 *>(input);
    half2 *output2 = reinterpret_cast<half2 *>(output);
    const dim3 block(h, head_num, 1);
    // limit h * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(h * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024 * 2));
    TransposeQkvKernel<half2><<<grid, block, 0, stream>>>(h, input2, output2);
  } else {
    const dim3 block(head_size, head_num, 1);
    // limit head_size * head_num to max block size(1024).
    PADDLE_ENFORCE_LE(head_size * head_num, 1024,
                      platform::errors::InvalidArgument(
                          "head_num (%d) * head_size (%d) should <= %d",
                          head_num, head_size, 1024));
    TransposeQkvKernel<half><<<grid, block, 0, stream>>>(head_size, input,
                                                         output);
  }
}
#endif

int QkvToContextPluginDynamic::initialize() { return 0; }

nvinfer1::DimsExprs QkvToContextPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  // input[0], (B, S, 3 * N * H, 1, 1)
  // input[1], (B, head_num, seq_len, seq_len)
  // output, (B, seq_len, hidden)
  PADDLE_ENFORCE_EQ(output_index, 0,
                    platform::errors::InvalidArgument(
                        "There is only one output of the EmbEltwiseLayernorm, "
                        "so the index should be zero,"
                        "but it's (%d)",
                        output_index));
  PADDLE_ENFORCE_EQ(
      nb_inputs, 2,
      platform::errors::InvalidArgument(
          "The Input of the EmbEltwiseLayernorm should be 3, but we found "
          "it has (%d) inputs",
          nb_inputs));
  nvinfer1::DimsExprs ret;
  ret.nbDims = 5;
  ret.d[0] = inputs[0].d[0];
  ret.d[1] = inputs[0].d[1];
  ret.d[2] = expr_builder.constant(head_size_ * head_number_);
  ret.d[3] = expr_builder.constant(1);
  ret.d[4] = expr_builder.constant(1);
  return ret;
}

bool QkvToContextPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    if (ban_fp16_) {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT ||
              in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType QkvToContextPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(
      index, 0, platform::errors::InvalidArgument(
                    "The EmbEltwiseLayernorm Plugin only has one input, so the "
                    "index value should be 0, but get %d.",
                    index));
  return input_types[0];
}

int QkvToContextPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  int input_num = ProductDim(input_dims);
  // input[0], (B, S, 3 * N * H, 1, 1)
  int batch = input_dims.d[0];
  int seq_len = input_dims.d[1];
  framework::Tensor multihead_temp_tensor;
  int scratch_size = batch * head_number_ * seq_len * seq_len * 1;

  int device_id;
  hipGetDevice(&device_id);
  multihead_temp_tensor.Resize({scratch_size + input_num});

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    auto *multihead_temp_data = multihead_temp_tensor.mutable_data<float>(
        platform::CUDAPlace(device_id));
    auto *qkptr = multihead_temp_data;
    auto *tptr = multihead_temp_data + scratch_size;

    const float *input0_data = static_cast<const float *>(inputs[0]);
    const float *input1_data = static_cast<const float *>(inputs[1]);
    // BxSx3xNxH => tptr: 3xBxNxSxH.
    TransposeQKV(batch, seq_len, head_size_, head_number_, input0_data, tptr,
                 stream);

    auto *device_ctx = static_cast<platform::CUDADeviceContext *>(
        platform::DeviceContextPool::Instance().Get(
            platform::CUDAPlace(device_id)));

    const platform::CUDADeviceContext &dev_ctx = *device_ctx;
    operators::math::MultiHeadGPUComputeFunctor<float> multihead_compute_func;
    multihead_compute_func(dev_ctx, batch, seq_len, head_number_, head_size_,
                           qkptr, input1_data, tptr, scale_,
                           static_cast<float>(0.0));

    int grid = batch * head_number_ * seq_len;
    int block = head_size_;
    float *output = static_cast<float *>(outputs[0]);
    transpose<float><<<grid, block, 0, stream>>>(tptr, output, batch, seq_len,
                                                 head_number_, head_size_);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    auto *multihead_temp_data =
        multihead_temp_tensor.mutable_data<int16_t>(  // NOLINT
            platform::CUDAPlace(device_id));

    half *qkptr = reinterpret_cast<half *>(multihead_temp_data);
    half *tptr = qkptr + scratch_size;

    const half *input0_data = static_cast<const half *>(inputs[0]);
    const half *input1_data = static_cast<const half *>(inputs[1]);
    // BxSx3xNxH => tptr: 3xBxNxSxH.
    TransposeQKV(batch, seq_len, head_size_, head_number_, input0_data, tptr,
                 stream);

    auto *device_ctx = static_cast<platform::CUDADeviceContext *>(
        platform::DeviceContextPool::Instance().Get(
            platform::CUDAPlace(device_id)));

    const platform::CUDADeviceContext &dev_ctx = *device_ctx;
    operators::math::MultiHeadGPUComputeFunctor<half> multihead_compute_func;
    multihead_compute_func(dev_ctx, batch, seq_len, head_number_, head_size_,
                           qkptr, input1_data, tptr, half(scale_), half(0.0));

    int grid = batch * head_number_ * seq_len;
    int block = head_size_;
    half *output = static_cast<half *>(outputs[0]);
    transpose<half><<<grid, block, 0, stream>>>(tptr, output, batch, seq_len,
                                                head_number_, head_size_);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The QKV TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
