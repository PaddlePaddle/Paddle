#include "hip/hip_runtime.h"
/*
Copyright (c) 2022, PaddlePaddle Authors, NVIDIA CORPORATION. All Rights
Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.  See
the License for the specific language governing permissions and
limitations under the License.
*/
#include "paddle/fluid/inference/tensorrt/plugin/spmm_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

nvinfer1::PluginFieldCollection SpmmPluginDynamicCreator::field_collection_{};
std::vector<nvinfer1::PluginField> SpmmPluginDynamicCreator::plugin_attr_;

inline int getElementSize(nvinfer1::DataType type) {
  switch (type) {
    case nvinfer1::DataType::kFLOAT:
      return 4;
    case nvinfer1::DataType::kHALF:
      return 2;
    case nvinfer1::DataType::kINT8:
      return 1;
    default:
      PADDLE_THROW(paddle::platform::errors::Fatal(
          "getElementSize only supports [FLOAT|HALF|INT8]"));
  }
}

inline hipDataType convertTrtType(nvinfer1::DataType type) {
  switch (type) {
    case nvinfer1::DataType::kFLOAT:
      return HIP_R_32F;
    case nvinfer1::DataType::kHALF:
      return HIP_R_16F;
    case nvinfer1::DataType::kINT8:
      return HIP_R_8I;
    default:
      PADDLE_THROW(paddle::platform::errors::Fatal(
          "getElementSize only supports [FLOAT|HALF|INT8]"));
  }
}

inline void deserialize_value_size(void const** buffer,
                                   size_t* buffer_size,
                                   void* value,
                                   size_t value_size) {
  PADDLE_ENFORCE_GE(
      *buffer_size,
      value_size,
      platform::errors::InvalidArgument("buffer_size must >= value_size"));
  memcpy(value, *buffer, value_size);
  reinterpret_cast<char const*&>(*buffer) += value_size;
  *buffer_size -= value_size;
}

inline float round_scale(float x) { return std::floor(x + 0.5f); }

inline void cudaFreeFunc(void* p) {
  if (p) {
    hipFree(p);
  }
}

inline void convertAndCopy(const nvinfer1::Weights& src,
                           nvinfer1::DataType type,
                           void* dest) {
  PADDLE_ENFORCE_EQ(src.type == nvinfer1::DataType::kFLOAT ||
                        src.type == nvinfer1::DataType::kHALF,
                    true,
                    platform::errors::InvalidArgument(
                        "convertAndCopy only supports src type [FLOAT|HALF]"));
  PADDLE_ENFORCE_EQ(
      type == nvinfer1::DataType::kFLOAT || type == nvinfer1::DataType::kHALF,
      true,
      platform::errors::InvalidArgument(
          "convertAndCopy only supports src type [FLOAT|HALF]"));

  if (type == nvinfer1::DataType::kFLOAT) {
    if (src.type == nvinfer1::DataType::kFLOAT) {
      std::copy_n(static_cast<const float*>(src.values),
                  src.count,
                  static_cast<float*>(dest));
    } else {
      for (int i = 0; i < src.count; ++i) {
        static_cast<float*>(dest)[i] =
            static_cast<float>(static_cast<const __half*>(src.values)[i]);
      }
    }
  } else {
    if (src.type == nvinfer1::DataType::kHALF) {
      std::copy_n(static_cast<const __half*>(src.values),
                  src.count,
                  static_cast<__half*>(dest));
    } else {
      for (int i = 0; i < src.count; ++i) {
        static_cast<__half*>(dest)[i] =
            static_cast<__half>(static_cast<const float*>(src.values)[i]);
      }
    }
  }
}

SpmmPluginDynamic::cusparseLtContext::cusparseLtContext() {
  paddle::platform::dynload::cusparseLtInit(&handle);
}

SpmmPluginDynamic::cusparseLtContext::~cusparseLtContext() {
  paddle::platform::dynload::cusparseLtDestroy(&handle);
}

void SpmmPluginDynamic::cusparseLtContext::init(
    int m,
    int n,
    int k,
    hipDataType type,
    void* bias_ptr,
    SpmmPluginDynamic::Activation activation) {
  /*
  1. Init matrix descriptors (matA, matB, matC)
  2. Init matrix multiplication descriptor (matmul)
  3. Set activation and bias attribute of matmul
  4. Init algorithm selection descriptor (alg_sel)
  5. Init plan descriptor (plan)
  */
  PADDLE_ENFORCE_EQ(
      is_initialized,
      false,
      platform::errors::InvalidArgument(
          "Descriptor should be destroyed before calling create"));
  constexpr int alignment = 16;
  cusparseComputeType compute_type;
  switch (type) {
    case HIP_R_32F:
      compute_type = CUSPARSE_COMPUTE_TF32;
      break;
    case HIP_R_16F:
      compute_type = CUSPARSE_COMPUTE_16F;
      break;
    case HIP_R_8I:
      compute_type = CUSPARSE_COMPUTE_32I;
      break;
    default:
      PADDLE_THROW(paddle::platform::errors::Fatal(
          "cusparLtContext only supports data type"
          "[HIP_R_32F|HIP_R_16F|HIP_R_8I]"));
  }
  paddle::platform::dynload::cusparseLtDenseDescriptorInit(
      &handle, &matA, m, k, k, alignment, type, HIPSPARSE_ORDER_ROW);
  paddle::platform::dynload::cusparseLtStructuredDescriptorInit(
      &handle,
      &matB,
      n,
      k,
      k,
      alignment,
      type,
      HIPSPARSE_ORDER_ROW,
      CUSPARSELT_SPARSITY_50_PERCENT);
  paddle::platform::dynload::cusparseLtDenseDescriptorInit(
      &handle, &matC, m, n, n, alignment, type, HIPSPARSE_ORDER_ROW);
  paddle::platform::dynload::cusparseLtMatmulDescriptorInit(
      &handle,
      &matmul,
      HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE,
      &matA,
      &matB,
      &matC,
      &matC,
      compute_type);
  if (activation == SpmmPluginDynamic::Activation::kRelu) {
    int true_value = 1;
    float relu_upper_bound = std::numeric_limits<float>::max();
    float relu_threshold = 0.0f;
    paddle::platform::dynload::cusparseLtMatmulDescSetAttribute(
        &handle,
        &matmul,
        CUSPARSELT_MATMUL_ACTIVATION_RELU,
        &true_value,
        sizeof(true_value));
    paddle::platform::dynload::cusparseLtMatmulDescSetAttribute(
        &handle,
        &matmul,
        CUSPARSELT_MATMUL_ACTIVATION_RELU_UPPERBOUND,
        &relu_upper_bound,
        sizeof(relu_upper_bound));
    paddle::platform::dynload::cusparseLtMatmulDescSetAttribute(
        &handle,
        &matmul,
        CUSPARSELT_MATMUL_ACTIVATION_RELU_THRESHOLD,
        &relu_threshold,
        sizeof(relu_threshold));
  } else if (activation == SpmmPluginDynamic::Activation::kGelu) {
    int true_value = 1;
    paddle::platform::dynload::cusparseLtMatmulDescSetAttribute(
        &handle,
        &matmul,
        CUSPARSELT_MATMUL_ACTIVATION_GELU,
        &true_value,
        sizeof(true_value));
  } else {
    PADDLE_ENFORCE_EQ(
        activation,
        SpmmPluginDynamic::Activation::kNone,
        platform::errors::InvalidArgument("Received unknown activation"));
  }
  if (bias_ptr != nullptr) {
    paddle::platform::dynload::cusparseLtMatmulDescSetAttribute(
        &handle,
        &matmul,
        CUSPARSELT_MATMUL_BIAS_POINTER,
        &bias_ptr,
        sizeof(bias_ptr));
  }
  paddle::platform::dynload::cusparseLtMatmulAlgSelectionInit(
      &handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT);
  int alg = 0;
  paddle::platform::dynload::cusparseLtMatmulAlgSetAttribute(
      &handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg, sizeof(alg));
  paddle::platform::dynload::cusparseLtMatmulGetWorkspace(
      &handle, &alg_sel, &workspace_size);
  paddle::platform::dynload::cusparseLtMatmulPlanInit(
      &handle, &plan, &matmul, &alg_sel, workspace_size);
  is_initialized = true;
}

void SpmmPluginDynamic::cusparseLtContext::setAlgo(int alg) {
  PADDLE_ENFORCE_EQ(
      is_initialized,
      true,
      platform::errors::InvalidArgument(
          "Descriptor should be initialized before setting algorithm"));
  paddle::platform::dynload::cusparseLtMatmulAlgSetAttribute(
      &handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg, sizeof(alg));
  paddle::platform::dynload::cusparseLtMatmulGetWorkspace(
      &handle, &alg_sel, &workspace_size);
  paddle::platform::dynload::cusparseLtMatmulPlanDestroy(&plan);
  paddle::platform::dynload::cusparseLtMatmulPlanInit(
      &handle, &plan, &matmul, &alg_sel, workspace_size);
}

void SpmmPluginDynamic::cusparseLtContext::destroy() {
  PADDLE_ENFORCE_EQ(is_initialized,
                    true,
                    platform::errors::InvalidArgument(
                        "cusparseLtContext is destroy before init"));
  paddle::platform::dynload::cusparseLtMatmulPlanDestroy(&plan);
  paddle::platform::dynload::cusparseLtMatDescriptorDestroy(&matC);
  paddle::platform::dynload::cusparseLtMatDescriptorDestroy(&matB);
  paddle::platform::dynload::cusparseLtMatDescriptorDestroy(&matA);
  is_initialized = false;
}

void SpmmPluginDynamic::cusparseLtContext::compressMatB(
    int n,
    int k,
    hipDataType type,
    void* src,
    void** dest,
    size_t* compressed_size) {
  PADDLE_ENFORCE_EQ(
      is_initialized,
      false,
      platform::errors::InvalidArgument(
          "cusparseLtContext should not initialized before compressMatB"));
  PADDLE_ENFORCE_EQ(*dest,
                    nullptr,
                    platform::errors::InvalidArgument(
                        "before compressMatB *dest must be nullptr"));
  constexpr int alignment = 16;
  paddle::platform::dynload::cusparseLtStructuredDescriptorInit(
      &handle,
      &matB,
      n,
      k,
      k,
      alignment,
      type,
      HIPSPARSE_ORDER_ROW,
      CUSPARSELT_SPARSITY_50_PERCENT);

  paddle::platform::dynload::cusparseLtSpMMACompressedSize2(
      &handle, &matB, compressed_size);
  hipMalloc(dest, *compressed_size);
  paddle::platform::dynload::cusparseLtSpMMACompress2(
      &handle, &matB, 0, HIPSPARSE_OPERATION_TRANSPOSE, src, *dest, nullptr);
  paddle::platform::dynload::cusparseLtMatDescriptorDestroy(&matB);
}

// Constructor for new plugin
SpmmPluginDynamic::SpmmPluginDynamic(const std::string& layer_name,
                                     const nvinfer1::DataType precision,
                                     const int out_dim,
                                     const nvinfer1::Weights& weight,
                                     const nvinfer1::Weights& bias,
                                     Activation activation)
    : layer_name_(layer_name),
      precision_(precision),
      out_dim_(out_dim),
      k_(0),
      m_max_(0),
      is_configured_(false),
      optim_alg_(0),
      weight_scale_(1.0f),
      weight_compressed_(nullptr),
      weight_compressed_dev_(nullptr),
      weight_compressed_dev_global_(nullptr),
      compressed_size_(0),
      has_bias_(false),
      bias_(nullptr),
      bias_dev_(nullptr),
      activation_(activation) {
  /*
  1. Convert weight precision (on host)
  2. (Int8) Calculate scale and scale the weight (on host)
  3. Copy weight to device
  4. Compress the weight (on device)
  5. Reset the shared_ptr "weight_compressed_dev_global_" to the compressed
  weight
  6. Copy the compressed weight to host
  7. Convert bias precision and copy (on host)
  */
  precision_size_ = getElementSize(precision);
  element_size_ =
      (precision_ == nvinfer1::DataType::kINT8 ? 4 : precision_size_);

  PADDLE_ENFORCE_EQ(
      weight.count % out_dim,
      0,
      platform::errors::InvalidArgument(
          "The size of weight should be divided by output dimension."));
  k_ = weight.count / out_dim;
  PADDLE_ENFORCE_EQ(
      weight.type == nvinfer1::DataType::kFLOAT ||
          weight.type == nvinfer1::DataType::kHALF,
      true,
      platform::errors::InvalidArgument(
          "SpmmPluginDynamic only supports weight of type [FLOAT|HALF]"));
  nvinfer1::DataType weight_type;
  if (precision_ == nvinfer1::DataType::kINT8) {
    weight_type = nvinfer1::DataType::kFLOAT;
  } else {
    weight_type = precision_;
  }
  std::vector<char> weight_host(element_size_ * out_dim_ * k_);
  convertAndCopy(weight, weight_type, weight_host.data());
  void* weight_dev{nullptr};
  hipMalloc(reinterpret_cast<void**>(&weight_dev),
             precision_size_ * out_dim_ * k_);
  if (precision == nvinfer1::DataType::kINT8) {
    float max_weight{0.0f};
    for (int i = 0; i < weight.count; ++i) {
      float local_abs =
          std::abs(reinterpret_cast<const float*>(weight_host.data())[i]);
      max_weight = std::max(max_weight, local_abs);
    }
    weight_scale_ = max_weight / 127.0f;
    std::vector<int8_t> scale_buffer(weight.count);
    for (int i = 0; i < weight.count; ++i) {
      scale_buffer[i] = static_cast<int8_t>(
          round_scale(reinterpret_cast<const float*>(weight_host.data())[i] /
                      weight_scale_));
    }
    hipMemcpy(weight_dev,
               scale_buffer.data(),
               precision_size_ * weight.count,
               hipMemcpyHostToDevice);
  } else {
    hipMemcpy(weight_dev,
               weight_host.data(),
               precision_size_ * weight.count,
               hipMemcpyHostToDevice);
  }
  spmm_context_.compressMatB(out_dim_,
                             k_,
                             convertTrtType(precision_),
                             weight_dev,
                             &weight_compressed_dev_,
                             &compressed_size_);
  weight_compressed_ = new char[compressed_size_];
  weight_compressed_dev_global_.reset(weight_compressed_dev_, cudaFreeFunc);
  hipMemcpy(weight_compressed_,
             weight_compressed_dev_global_.get(),
             compressed_size_,
             hipMemcpyDeviceToHost);
  has_bias_ = (bias.count != 0);
  if (has_bias_) {
    if (bias.count != out_dim) {
      PADDLE_THROW(paddle::platform::errors::Fatal(
          "The dimension of bias should be equal to output dimension"));
    }
    if (precision_ == nvinfer1::DataType::kHALF) {
      bias_ = new half[out_dim_];
      convertAndCopy(bias, nvinfer1::DataType::kHALF, bias_);
    } else {
      bias_ = new float[out_dim_];
      convertAndCopy(bias, nvinfer1::DataType::kFLOAT, bias_);
    }
  }

  hipFree(weight_dev);
}

// Constructor for clone
SpmmPluginDynamic::SpmmPluginDynamic(const std::string& layer_name,
                                     const nvinfer1::DataType precision,
                                     const int out_dim,
                                     const int k,
                                     const void* weight_compressed,
                                     size_t compressed_size,
                                     const void* bias,
                                     bool is_configured,
                                     const int m_max,
                                     const int optim_alg,
                                     Activation activation)
    : layer_name_(layer_name),
      precision_(precision),
      out_dim_(out_dim),
      k_(k),
      m_max_(m_max),
      is_configured_(is_configured),
      optim_alg_(optim_alg),
      weight_scale_(1.0f),
      weight_compressed_(nullptr),
      weight_compressed_dev_global_(nullptr),
      compressed_size_(compressed_size),
      has_bias_(false),
      bias_(nullptr),
      bias_dev_(nullptr),
      activation_(activation) {
  /*
  1. Copy the compressed weight (on host)
  2. Copy the bias (on host)
  3. (Configured) Copy the bias to device
  4. (Configured) Init cuSPARSELt descriptors
  */
  precision_size_ = getElementSize(precision);
  element_size_ =
      (precision_ == nvinfer1::DataType::kINT8 ? 4 : precision_size_);
  // Each plugin has a copy of compressed weight on host, while sharing the
  // compressed weights on device using std::shared_ptr
  weight_compressed_ = new char[compressed_size];
  std::copy_n(static_cast<const char*>(weight_compressed),
              compressed_size,
              static_cast<char*>(weight_compressed_));

  has_bias_ = (bias != nullptr);
  if (has_bias_) {
    // Each plugin has a copy of bias
    bias_ = new float[out_dim_];
    std::copy_n(static_cast<const char*>(bias),
                sizeof(float) * out_dim_,
                static_cast<char*>(bias_));
    if (is_configured_) {
      hipMalloc(reinterpret_cast<void**>(&bias_dev_),
                 sizeof(float) * out_dim_);
      hipMemcpy(
          bias_dev_, bias_, sizeof(float) * out_dim_, hipMemcpyHostToDevice);
    }
  }

  if (is_configured_) {
    hipDataType dataType = convertTrtType(precision_);
    spmm_context_.init(m_max_, out_dim_, k_, dataType, bias_dev_, activation_);
    spmm_context_.setAlgo(optim_alg_);
  }
}

SpmmPluginDynamic::SpmmPluginDynamic(const std::string name,
                                     const void* data,
                                     size_t length)
    : layer_name_(name),
      weight_compressed_(nullptr),
      weight_compressed_dev_(nullptr),
      weight_compressed_dev_global_(nullptr),
      bias_(nullptr),
      bias_dev_(nullptr) {
  DeserializeValue(&data, &length, &precision_);
  DeserializeValue(&data, &length, &precision_size_);
  DeserializeValue(&data, &length, &element_size_);
  DeserializeValue(&data, &length, &out_dim_);
  DeserializeValue(&data, &length, &k_);
  DeserializeValue(&data, &length, &m_max_);
  DeserializeValue(&data, &length, &is_configured_);
  DeserializeValue(&data, &length, &optim_alg_);
  DeserializeValue(&data, &length, &weight_scale_);
  DeserializeValue(&data, &length, &compressed_size_);
  DeserializeValue(&data, &length, &has_bias_);
  DeserializeValue(&data, &length, &activation_);

  PADDLE_ENFORCE_EQ(is_configured_,
                    true,
                    platform::errors::InvalidArgument(
                        "Deserialize data should be configured"));
  weight_compressed_ = new char[compressed_size_];
  deserialize_value_size(&data, &length, weight_compressed_, compressed_size_);
  hipMalloc(reinterpret_cast<void**>(&weight_compressed_dev_),
             compressed_size_);
  hipMemcpy(weight_compressed_dev_,
             weight_compressed_,
             compressed_size_,
             hipMemcpyHostToDevice);
  weight_compressed_dev_global_.reset(weight_compressed_dev_, cudaFreeFunc);

  if (has_bias_) {
    bias_ = new float[out_dim_];
    deserialize_value_size(&data, &length, bias_, sizeof(float) * out_dim_);
    hipMalloc(reinterpret_cast<void**>(&bias_dev_), sizeof(float) * out_dim_);
    hipMemcpy(
        bias_dev_, bias_, sizeof(float) * out_dim_, hipMemcpyHostToDevice);
  }

  if (is_configured_) {
    hipDataType dataType = convertTrtType(precision_);
    spmm_context_.init(m_max_, out_dim_, k_, dataType, bias_dev_, activation_);
    spmm_context_.setAlgo(optim_alg_);
  }
}

nvinfer1::IPluginV2DynamicExt* SpmmPluginDynamic::clone() const noexcept {
  try {
    auto* p = new SpmmPluginDynamic(layer_name_,
                                    precision_,
                                    out_dim_,
                                    k_,
                                    weight_compressed_,
                                    compressed_size_,
                                    bias_,
                                    is_configured_,
                                    m_max_,
                                    optim_alg_,
                                    activation_);
    p->weight_scale_ = weight_scale_;
    p->weight_compressed_dev_global_ = weight_compressed_dev_global_;
    p->setPluginNamespace(namespace_.c_str());
    return p;
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
  return nullptr;
}

nvinfer1::DimsExprs SpmmPluginDynamic::getOutputDimensions(
    int outputIndex,
    const nvinfer1::DimsExprs* inputs,
    int nbInputs,
    nvinfer1::IExprBuilder& exprBuilder) noexcept {
  int nbDims = inputs[0].nbDims;
  try {
    PADDLE_ENFORCE_EQ(nbInputs,
                      1,
                      platform::errors::InvalidArgument(
                          "SpmmPluginDynamic's nbInputs is invalid"));
    PADDLE_ENFORCE_EQ(outputIndex,
                      0,
                      platform::errors::InvalidArgument(
                          "SpmmPluginDynamic's outputIndex is invalid"));
    if (nbDims == 5) {
      int nbDims = inputs[0].nbDims;
      PADDLE_ENFORCE_EQ(
          inputs[0].d[3]->getConstantValue(),
          1,
          platform::errors::InvalidArgument("now the input d[3] should be 1"));
      PADDLE_ENFORCE_EQ(
          inputs[0].d[4]->getConstantValue(),
          1,
          platform::errors::InvalidArgument("now the input d[4] should be 1"));
      nvinfer1::DimsExprs ret;
      ret.nbDims = nbDims;
      ret.d[0] = inputs[0].d[0];
      ret.d[1] = inputs[0].d[1];
      ret.d[2] = exprBuilder.constant(out_dim_);
      ret.d[3] = exprBuilder.constant(1);
      ret.d[4] = exprBuilder.constant(1);
      return ret;
    } else if (nbDims == 4) {
      int nbDims = inputs[0].nbDims;
      PADDLE_ENFORCE_EQ(
          inputs[0].d[2]->getConstantValue(),
          1,
          platform::errors::InvalidArgument("now the input d[2] should be 1"));
      PADDLE_ENFORCE_EQ(
          inputs[0].d[3]->getConstantValue(),
          1,
          platform::errors::InvalidArgument("now the input d[3] should be 1"));
      nvinfer1::DimsExprs ret;
      ret.nbDims = nbDims;
      ret.d[0] = inputs[0].d[0];
      ret.d[1] = exprBuilder.constant(out_dim_);
      ret.d[2] = exprBuilder.constant(1);
      ret.d[3] = exprBuilder.constant(1);

      return ret;
    } else {
      PADDLE_THROW(paddle::platform::errors::Fatal("nbDims should be 4 or 5"));
    }
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
  return nvinfer1::DimsExprs{};
}

bool SpmmPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc* inOut,
    int nbInputs,
    int nbOutputs) noexcept {
  PADDLE_ENFORCE_EQ(nbInputs,
                    1,
                    platform::errors::InvalidArgument(
                        "SpmmPluginDynamic's nbInputs should be 1"));
  PADDLE_ENFORCE_EQ(nbOutputs,
                    1,
                    platform::errors::InvalidArgument(
                        "SpmmPluginDynamic's nbOutputs should be 1"));

  const nvinfer1::PluginTensorDesc& in = inOut[pos];
  if (pos == 0) {
    return (in.type == precision_) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
  }
  const nvinfer1::PluginTensorDesc& prev = inOut[pos - 1];

  return in.type == prev.type && in.format == prev.format;
}

void SpmmPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* inputs,
    int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* outputs,
    int nbOutputs) noexcept {
  /*
  The following steps are executed if not configured.
  1. (INT8) Scale the bias (on host)
  2. Copy the bias to device
  3. Search the optimal algorithm
  */
  try {
    PADDLE_ENFORCE_EQ(nbInputs,
                      1,
                      platform::errors::InvalidArgument(
                          "SpmmPluginDynamic's nbInputs should be 1"));
    PADDLE_ENFORCE_EQ(nbOutputs,
                      1,
                      platform::errors::InvalidArgument(
                          "SpmmPluginDynamic's nbOutputs should be 1"));
    PADDLE_ENFORCE_EQ(precision_,
                      inputs[0].desc.type,
                      platform::errors::InvalidArgument(
                          "precision_ should be equal to inputs[0].desc.type"));
    const auto& inDims0 = inputs[0].desc.dims;
    if (inDims0.nbDims == 5) {
      PADDLE_ENFORCE_EQ(
          inDims0.nbDims,
          5,
          platform::errors::InvalidArgument("inDims0.nbDims should be 5"));
      PADDLE_ENFORCE_EQ(k_,
                        inDims0.d[2],
                        platform::errors::InvalidArgument(
                            "inDims0.d[2] should be equals to k"));
      PADDLE_ENFORCE_EQ(
          inDims0.d[3],
          1,
          platform::errors::InvalidArgument("inDims0.d[3] should be 1"));
      PADDLE_ENFORCE_EQ(
          inDims0.d[4],
          1,
          platform::errors::InvalidArgument("inDims0.d[4] should be 1"));
      const int BS = inputs->max.d[0];
      const int Seq = inputs->max.d[1];
      m_max_ = BS * Seq;
    } else if (inDims0.nbDims == 4) {
      PADDLE_ENFORCE_EQ(
          inDims0.nbDims,
          4,
          platform::errors::InvalidArgument("inDims0.nbDims should be 4"));
      PADDLE_ENFORCE_EQ(k_,
                        inDims0.d[1],
                        platform::errors::InvalidArgument(
                            "inDims0.d[1] should be equals to k"));
      PADDLE_ENFORCE_EQ(
          inDims0.d[2],
          1,
          platform::errors::InvalidArgument("inDims0.d[2] should be 1"));
      PADDLE_ENFORCE_EQ(
          inDims0.d[3],
          1,
          platform::errors::InvalidArgument("inDims0.d[3] should be 1"));
      const int BS_Seq = inputs->max.d[0];
      m_max_ = BS_Seq;
    }
    if (is_configured_) {
      return;
    }

    if (has_bias_) {
      if (inputs->desc.type == nvinfer1::DataType::kINT8) {
        for (int i = 0; i < out_dim_; ++i) {
          static_cast<float*>(bias_)[i] =
              static_cast<const float*>(bias_)[i] / outputs->desc.scale;
        }
      }
      hipMalloc(reinterpret_cast<void**>(&bias_dev_),
                 sizeof(float) * out_dim_);
      hipMemcpy(
          bias_dev_, bias_, sizeof(float) * out_dim_, hipMemcpyHostToDevice);
    }
    hipDataType dataType = convertTrtType(precision_);
    spmm_context_.init(m_max_, out_dim_, k_, dataType, bias_dev_, activation_);

    void* dA;
    void* dC;
    void* d_workspace;
    float alpha{1.0f};
    float beta{0.0f};
    if (precision_ == nvinfer1::DataType::kINT8) {
      alpha = inputs->desc.scale * weight_scale_ / outputs->desc.scale;
    }
    hipMalloc(reinterpret_cast<void**>(&dA), m_max_ * k_ * sizeof(dataType));
    hipMalloc(reinterpret_cast<void**>(&dC),
               m_max_ * out_dim_ * sizeof(dataType));
    hipMalloc(reinterpret_cast<void**>(&d_workspace),
               spmm_context_.workspace_size);
    paddle::platform::dynload::cusparseLtMatmulSearch(
        &spmm_context_.handle,
        &spmm_context_.plan,
        &alpha,
        dA,
        weight_compressed_dev_global_.get(),
        &beta,
        dC,
        dC,
        d_workspace,
        nullptr,
        0);
    paddle::platform::dynload::cusparseLtMatmulAlgGetAttribute(
        &spmm_context_.handle,
        &spmm_context_.alg_sel,
        CUSPARSELT_MATMUL_ALG_CONFIG_ID,
        &optim_alg_,
        sizeof(optim_alg_));
    hipFree(dA);
    hipFree(dC);
    hipFree(d_workspace);

    is_configured_ = true;
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
}

size_t SpmmPluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs,
    int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs,
    int nbOutputs) const noexcept {
  return spmm_context_.workspace_size;
}

int SpmmPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
                               const nvinfer1::PluginTensorDesc* outputDesc,
                               const void* const* inputs,
                               void* const* outputs,
                               void* workSpace,
                               hipStream_t stream) noexcept {
  try {
    PADDLE_ENFORCE_EQ(is_configured_,
                      true,
                      platform::errors::InvalidArgument(
                          "The plugin is not configured before enqueue"));
    if (inputDesc->dims.nbDims == 5) {
      PADDLE_ENFORCE_EQ(
          k_,
          inputDesc->dims.d[2],
          platform::errors::InvalidArgument("k_ == inputDesc->dims.d[2]"));
    } else if (inputDesc->dims.nbDims == 4) {
      PADDLE_ENFORCE_EQ(
          k_,
          inputDesc->dims.d[1],
          platform::errors::InvalidArgument("k_ == inputDesc->dims.d[1]"));
    }
    float alpha = 1.0f;
    float beta = 0.0f;
    if (inputDesc->type == nvinfer1::DataType::kFLOAT) {
      const auto* const input = static_cast<const float*>(inputs[0]);
      auto* output = static_cast<float*>(outputs[0]);
      auto* weight_compressed_dev_p_ = weight_compressed_dev_global_.get();
      hipsparseStatus_t status =
          paddle::platform::dynload::cusparseLtMatmul(&spmm_context_.handle,
                                                      &spmm_context_.plan,
                                                      &alpha,
                                                      input,
                                                      weight_compressed_dev_p_,
                                                      &beta,
                                                      output,
                                                      output,
                                                      workSpace,
                                                      &stream,
                                                      1);
      return status != HIPSPARSE_STATUS_SUCCESS;
    } else if (inputDesc->type == nvinfer1::DataType::kHALF) {
      const auto* const input = static_cast<const half*>(inputs[0]);
      auto* output = static_cast<half*>(outputs[0]);
      auto* weight_compressed_dev_p_ = weight_compressed_dev_global_.get();
      hipsparseStatus_t status =
          paddle::platform::dynload::cusparseLtMatmul(&spmm_context_.handle,
                                                      &spmm_context_.plan,
                                                      &alpha,
                                                      input,
                                                      weight_compressed_dev_p_,
                                                      &beta,
                                                      output,
                                                      output,
                                                      workSpace,
                                                      &stream,
                                                      1);
      return status != HIPSPARSE_STATUS_SUCCESS;
    } else if (inputDesc->type == nvinfer1::DataType::kINT8) {
      alpha = inputDesc->scale * weight_scale_ / outputDesc->scale;
      const auto* const input = static_cast<const int8_t*>(inputs[0]);
      auto* output = static_cast<int8_t*>(outputs[0]);
      auto* weight_compressed_dev_p_ = weight_compressed_dev_global_.get();
      hipsparseStatus_t status =
          paddle::platform::dynload::cusparseLtMatmul(&spmm_context_.handle,
                                                      &spmm_context_.plan,
                                                      &alpha,
                                                      input,
                                                      weight_compressed_dev_p_,
                                                      &beta,
                                                      output,
                                                      output,
                                                      workSpace,
                                                      &stream,
                                                      1);
      return status != HIPSPARSE_STATUS_SUCCESS;
    } else {
      PADDLE_THROW(paddle::platform::errors::Fatal(
          "Unsupported type error, expected [kHALF,kFLOAT], but received %d",
          static_cast<int>(precision_)));
    }
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
  return -1;
}

nvinfer1::DataType SpmmPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType* inputTypes,
    int nbInputs) const noexcept {
  PADDLE_ENFORCE_EQ(index,
                    0,
                    platform::errors::InvalidArgument(
                        "SpmmPluginDynamic's index should be 0"));
  PADDLE_ENFORCE_EQ(nbInputs,
                    1,
                    platform::errors::InvalidArgument(
                        "SpmmPluginDynamic's nbInputs should be 1"));
  PADDLE_ENFORCE_EQ(inputTypes[0] == nvinfer1::DataType::kFLOAT ||
                        inputTypes[0] == nvinfer1::DataType::kHALF ||
                        inputTypes[0] == nvinfer1::DataType::kINT8,
                    true,
                    platform::errors::InvalidArgument(
                        "SpmmPluginDynamic is not support this format now"));

  return inputTypes[0];
}

const char* SpmmPluginDynamic::getPluginType() const noexcept {
  return "SpmmPluginDynamic";
}

const char* SpmmPluginDynamic::getPluginVersion() const noexcept { return "1"; }

int SpmmPluginDynamic::getNbOutputs() const noexcept { return 1; }

int SpmmPluginDynamic::initialize() noexcept { return 0; }

void SpmmPluginDynamic::terminate() noexcept {}

size_t SpmmPluginDynamic::getSerializationSize() const noexcept {
  return compressed_size_ + (has_bias_ ? sizeof(float) * out_dim_ : 0) +
         sizeof(precision_) + sizeof(precision_size_) + sizeof(element_size_) +
         sizeof(out_dim_) + sizeof(k_) + sizeof(m_max_) +
         sizeof(is_configured_) + sizeof(optim_alg_) + sizeof(weight_scale_) +
         sizeof(compressed_size_) + sizeof(has_bias_) + sizeof(activation_);
}

void SpmmPluginDynamic::serialize(void* buffer) const noexcept {
  SerializeValue(&buffer, precision_);
  SerializeValue(&buffer, precision_size_);
  SerializeValue(&buffer, element_size_);
  SerializeValue(&buffer, out_dim_);
  SerializeValue(&buffer, k_);
  SerializeValue(&buffer, m_max_);
  SerializeValue(&buffer, is_configured_);
  SerializeValue(&buffer, optim_alg_);
  SerializeValue(&buffer, weight_scale_);
  SerializeValue(&buffer, compressed_size_);
  SerializeValue(&buffer, has_bias_);
  SerializeValue(&buffer, activation_);
  char* d = static_cast<char*>(buffer);
  std::copy_n(
      static_cast<const char*>(weight_compressed_), compressed_size_, d);
  if (has_bias_) {
    d += compressed_size_;
    std::copy_n(static_cast<const char*>(bias_), out_dim_ * sizeof(float), d);
  }
}

void SpmmPluginDynamic::destroy() noexcept {
  delete[] reinterpret_cast<char*>(weight_compressed_);
  if (has_bias_) {
    hipFree(bias_dev_);
  }
  if (is_configured_) {
    spmm_context_.destroy();
  }
  delete this;
}

void SpmmPluginDynamic::setPluginNamespace(const char* libNamespace) noexcept {
  try {
    namespace_ = libNamespace;
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
}

const char* SpmmPluginDynamic::getPluginNamespace() const noexcept {
  return namespace_.c_str();
}

inline nvinfer1::DataType fieldTypeToDataType(
    const nvinfer1::PluginFieldType ftype) {
  switch (ftype) {
    case nvinfer1::PluginFieldType::kFLOAT32:
      return nvinfer1::DataType::kFLOAT;
    case nvinfer1::PluginFieldType::kFLOAT16:
      return nvinfer1::DataType::kHALF;
    case nvinfer1::PluginFieldType::kINT32:
      return nvinfer1::DataType::kINT32;
    case nvinfer1::PluginFieldType::kINT8:
      return nvinfer1::DataType::kINT8;
    default:
      PADDLE_THROW(paddle::platform::errors::Fatal(
          "No corresponding datatype for plugin field type"));
  }
}

SpmmPluginDynamicCreator::SpmmPluginDynamicCreator() {
  plugin_attr_.emplace_back(nvinfer1::PluginField(
      "type_id", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
  plugin_attr_.emplace_back(nvinfer1::PluginField(
      "out_dim", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
  plugin_attr_.emplace_back(nvinfer1::PluginField(
      "weight", nullptr, nvinfer1::PluginFieldType::kFLOAT32, 1));
  plugin_attr_.emplace_back(nvinfer1::PluginField(
      "bias", nullptr, nvinfer1::PluginFieldType::kFLOAT32, 1));
  plugin_attr_.emplace_back(nvinfer1::PluginField(
      "activation_id", nullptr, nvinfer1::PluginFieldType::kINT8, 1));

  field_collection_.nbFields = plugin_attr_.size();
  field_collection_.fields = plugin_attr_.data();
}

const char* SpmmPluginDynamicCreator::getPluginName() const noexcept {
  return "SpmmPluginDynamic";
}

const char* SpmmPluginDynamicCreator::getPluginVersion() const noexcept {
  return "1";
}

const nvinfer1::PluginFieldCollection*
SpmmPluginDynamicCreator::getFieldNames() noexcept {
  return &field_collection_;
}

nvinfer1::IPluginV2* SpmmPluginDynamicCreator::createPlugin(
    const char* name, const nvinfer1::PluginFieldCollection* fc) noexcept {
  try {
    int type_id = -1;
    int out_dim = 0;
    nvinfer1::Weights weight{nvinfer1::DataType::kFLOAT, nullptr, 0ll};
    nvinfer1::Weights bias{nvinfer1::DataType::kFLOAT, nullptr, 0ll};
    int activation_id = -1;

    for (int i = 0; i < fc->nbFields; i++) {
      std::string field_name(fc->fields[i].name);
      if (field_name.compare("type_id") == 0) {
        type_id = static_cast<const int*>(fc->fields[i].data)[0];
      } else if (field_name.compare("out_dim") == 0) {
        out_dim = static_cast<const int*>(fc->fields[i].data)[0];
      } else if (field_name.compare("weight") == 0) {
        weight.type = fieldTypeToDataType(fc->fields[i].type);
        weight.values = fc->fields[i].data;
        weight.count = fc->fields[i].length;
      } else if (field_name.compare("bias") == 0) {
        bias.type = fieldTypeToDataType(fc->fields[i].type);
        bias.values = fc->fields[i].data;
        bias.count = fc->fields[i].length;
      } else if (field_name.compare("activation_id") == 0) {
        activation_id = static_cast<const int*>(fc->fields[i].data)[0];
      } else {
        PADDLE_THROW(paddle::platform::errors::Fatal("Unsupport plugin field"));
      }
    }

    PADDLE_ENFORCE_NE(
        type_id,
        -1,
        platform::errors::InvalidArgument(
            "SpmmPluginDynamicCreator's type_id should not be -1"));
    PADDLE_ENFORCE_NE(
        out_dim,
        0,
        platform::errors::InvalidArgument(
            "SpmmPluginDynamicCreator's out_dim should not be 0"));
    PADDLE_ENFORCE_NE(
        weight.count,
        0,
        platform::errors::InvalidArgument(
            "SpmmPluginDynamicCreator's weight size should not be 0"));
    PADDLE_ENFORCE_NE(
        activation_id,
        -1,
        platform::errors::InvalidArgument(
            "SpmmPluginDynamicCreator's activation_id should not be -1"));
    nvinfer1::DataType type = static_cast<nvinfer1::DataType>(type_id);
    SpmmPluginDynamic::Activation activation =
        static_cast<SpmmPluginDynamic::Activation>(activation_id);
    return new SpmmPluginDynamic(name, type, out_dim, weight, bias, activation);
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
  return nullptr;
}

nvinfer1::IPluginV2* SpmmPluginDynamicCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength) noexcept {
  // This object will be deleted when the network is destroyed, which will
  // call SpmmPluginDynamic::destroy()
  try {
    return new SpmmPluginDynamic(name, serialData, serialLength);
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
  return nullptr;
}

void SpmmPluginDynamicCreator::setPluginNamespace(
    const char* libNamespace) noexcept {
  try {
    namespace_ = libNamespace;
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
  }
}

const char* SpmmPluginDynamicCreator::getPluginNamespace() const noexcept {
  return namespace_.c_str();
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
