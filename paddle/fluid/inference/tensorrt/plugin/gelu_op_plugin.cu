#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include <vector>
#include "paddle/fluid/inference/tensorrt/plugin/gelu_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// constants for approximating the normal cdf
constexpr float A = 1.41421356237309504;  // sqrt(2)

GeluPlugin* CreateGeluPluginDeserialize(const void* buffer, size_t length) {
  return new GeluPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("gelu plugin", CreateGeluPluginDeserialize);

nvinfer1::Dims GeluPlugin::getOutputDimensions(int index,
                                               const nvinfer1::Dims* in_dims,
                                               int nb_inputs) {
  assert(nb_inputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const& input_dims = in_dims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

template <typename T, unsigned TPB>
__global__ void geluKernel(const T a, int n, const T* input, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;
  if (idx < n) {
    const T in = input[idx];
    const T cdf = 0.5 * (1.0 + erf(in * 0.5 * a));
    output[idx] = in * cdf;
  }
}

int computeGelu(hipStream_t stream, int n, const float* input, float* output) {
  constexpr int blockSize = 256;
  const int gridSize = (n + blockSize - 1) / blockSize;
  geluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, n, input,
                                                                   output);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) LOG(ERROR) << hipGetErrorString(error);
  return 0;
}

int GeluPlugin::enqueue(int batchSize, const void* const* inputs,
                        void** outputs, void*, hipStream_t stream) {
  int status = -1;
  const float* input = static_cast<const float*>(inputs[0]);
  float* output = static_cast<float*>(outputs[0]);
  status = computeGelu(stream, input_volume_ * batchSize, input, output);
  return status;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
