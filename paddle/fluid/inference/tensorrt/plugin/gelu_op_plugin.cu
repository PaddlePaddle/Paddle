#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include <vector>
#include "paddle/fluid/inference/tensorrt/plugin/gelu_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// constants for approximating the normal cdf
constexpr float A = 0.5;

constexpr float B = 0.7978845608028654;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125;  // 0.044715 * sqrt(2.0/M_PI)

GeluPlugin* CreateGeluPluginDeserialize(const void* buffer, size_t length) {
  return new GeluPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("gelu plugin", CreateGeluPluginDeserialize);

nvinfer1::Dims GeluPlugin::getOutputDimensions(int index,
                                               const nvinfer1::Dims* in_dims,
                                               int nb_inputs) {
  assert(nb_inputs == 1);
  assert(index < this->getNbOutputs());
  nvinfer1::Dims const& input_dims = in_dims[0];
  nvinfer1::Dims output_dims = input_dims;
  return output_dims;
}

template <typename T, unsigned TPB>
__global__ void geluKernel(const T a, const T b, const T c, int n,
                           const T* input, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < n) {
    const T in = input[idx];
    const T cdf = a + a * tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

int computeGelu(hipStream_t stream, int n, const float* input, float* output) {
  constexpr int blockSize = 256;
  const int gridSize = (n + blockSize - 1) / blockSize;
  geluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(
      A, B, C, n, input, output);

  CHECK(hipPeekAtLastError());
  return 0;
}

int GeluPlugin::enqueue(int batchSize, const void* const* inputs,
                        void** outputs, void*, hipStream_t stream) {
  int status = -1;

  const float* input = static_cast<const float*>(inputs[0]);
  float* output = static_cast<float*>(outputs[0]);
  status = computeGelu(stream, input_volume_ * batchSize, input, output);

  return status;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
