#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/slice_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

template <typename T>
__global__ void SliceKernel(int num, int dims, const T *input,
                            const int *offsets_info, T *output) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int shared_data[];

  if (threadIdx.x == 0) {
    for (int i = 0; i < dims * 3; i++) {
      shared_data[i] = offsets_info[i];
    }
  }
  __syncthreads();

  if (idx < num) {
    int t_idx = idx;
    int in_idx = 0;
    for (int i = dims - 1; i >= 0; i--) {
      // output_shape
      auto t = t_idx % shared_data[i * 3 + 1];
      // out offset
      auto s = t + shared_data[i * 3];
      // input_seg_offset
      in_idx = in_idx + shared_data[i * 3 + 2] * s;
      t_idx = t_idx / shared_data[i * 3 + 1];
    }
    output[idx] = input[in_idx];
  }
}

int SlicePluginDynamic::initialize() { return 0; }

size_t SlicePluginDynamic::getSerializationSize() const { return 0; }

void SlicePluginDynamic::serialize(void *buffer) const {}

nvinfer1::DimsExprs SlicePluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  auto in_dims = inputs[0];
  nvinfer1::DimsExprs ret = in_dims;
  // start, ends should greater 0
  for (size_t i = 0; i < axes_.size(); i++) {
    int start = starts_[i];
    int end = ends_[i];
    ret.d[axes_[i]] = expr_builder.constant(end - start);
  }
  return ret;
}

bool SlicePluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
#ifdef SUPPORTS_CUDA_FP16
    if (ban_fp16_) {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT ||
              in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
#else
    return (in.type == nvinfer1::DataType::kFLOAT) &&
           (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType SlicePluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The Slice Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT ||
                     input_types[0] == nvinfer1::DataType::kHALF),
                    true, platform::errors::InvalidArgument(
                              "The input type should be half or float"));
  return input_types[0];
}

int SlicePluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                                const nvinfer1::PluginTensorDesc *output_desc,
                                const void *const *inputs, void *const *outputs,
                                void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  auto out_dims = output_desc[0].dims;
  auto num_dims = input_dims.nbDims;
  size_t out_num = ProductDim(out_dims);

  std::vector<int> seg_offsets;
  std::vector<int> offsets;
  std::vector<int> extends;

  offsets.reserve(num_dims);
  extends.reserve(num_dims);
  seg_offsets.reserve(num_dims);

  seg_offsets[num_dims - 1] = 1;
  for (int i = num_dims - 2; i >= 0; i--) {
    seg_offsets[i] = input_dims.d[i + 1] * seg_offsets[i + 1];
  }

  for (size_t i = 0; i < num_dims; ++i) {
    offsets[i] = 0;
    extends[i] = out_dims.d[i];
  }
  for (size_t i = 0; i < axes_.size(); ++i) {
    offsets[axes_[i]] = starts_[i];
  }

  std::vector<int> offset_info;
  for (size_t i = 0; i < num_dims; ++i) {
    offset_info.push_back(offsets[i]);
    offset_info.push_back(extends[i]);
    offset_info.push_back(seg_offsets[i]);
  }

  framework::Tensor offset_temp_tensor;

  int device_id;
  hipGetDevice(&device_id);
  offset_temp_tensor.Resize({3 * num_dims});
  auto *offset_temp_data =
      offset_temp_tensor.mutable_data<int>(platform::CUDAPlace(device_id));

  hipMemcpyAsync(offset_temp_data, offset_info.data(),
                  sizeof(int) * 3 * num_dims, hipMemcpyHostToDevice, stream);

  int threads = 256;
  int blocks = (out_num + threads - 1) / threads;
  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    const float *input1 = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    SliceKernel<float><<<blocks, threads, 3 * num_dims * sizeof(int), stream>>>(
        out_num, num_dims, input1, offset_temp_data, output);
  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef SUPPORTS_CUDA_FP16
    const half *input1 = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    SliceKernel<half><<<blocks, threads, 3 * num_dims * sizeof(int), stream>>>(
        out_num, num_dims, input1, offset_temp_data, output);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The cuda archs you specific should greater than 600."));
#endif
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The Slice TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
