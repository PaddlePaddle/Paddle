#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/slice_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

SlicePlugin *CreateSlicePluginDeserialize(const void *buffer, size_t length) {
  return new SlicePlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("slice_plugin", CreateSlicePluginDeserialize);

template <typename T>
__global__ void SliceKernel(int num, int dims, const T *input,
                            const int *offsets_info, T *output) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int shared_data[];

  if (threadIdx.x == 0) {
    for (int i = 0; i < dims * 3; i++) {
      shared_data[i] = offsets_info[i];
    }
  }
  __syncthreads();

  if (idx < num) {
    int t_idx = idx;
    int in_idx = 0;
    for (int i = dims - 1; i >= 0; i--) {
      // output_shape
      auto t = t_idx % shared_data[i * 3 + 1];
      // out offset
      auto s = t + shared_data[i * 3];
      // input_seg_offset
      in_idx = in_idx + shared_data[i * 3 + 2] * s;
      t_idx = t_idx / shared_data[i * 3 + 1];
    }
    output[idx] = input[in_idx];
  }
}

SlicePlugin::SlicePlugin(std::vector<int> starts, std::vector<int> ends,
                         std::vector<int> axes, bool with_fp16)
    : starts_(starts), ends_(ends), axes_(axes) {
  with_fp16_ = with_fp16;
  hipEventCreate(&copy_event_);
  hipStreamCreate(&copy_stream_);
}

SlicePlugin::SlicePlugin(void const *serial_data, size_t serial_length) {
  deserializeBase(serial_data, serial_length);
  DeserializeValue(&serial_data, &serial_length, &starts_);
  DeserializeValue(&serial_data, &serial_length, &ends_);
  DeserializeValue(&serial_data, &serial_length, &axes_);
  hipEventCreate(&copy_event_);
  hipStreamCreate(&copy_stream_);
}

SlicePlugin::~SlicePlugin() {
  hipStreamDestroy(copy_stream_);
  hipEventDestroy(copy_event_);
  hipFree(offset_temp_data_);
}

SlicePlugin *SlicePlugin::clone() const {
  return new SlicePlugin(starts_, ends_, axes_, with_fp16_);
}

bool SlicePlugin::supportsFormat(nvinfer1::DataType type,
                                 nvinfer1::PluginFormat format) const {
  if (with_fp16_) {
    return ((type == nvinfer1::DataType::kFLOAT ||
             type == nvinfer1::DataType::kHALF) &&
            (format == nvinfer1::PluginFormat::kNCHW));
  } else {
    return ((type == nvinfer1::DataType::kFLOAT) &&
            (format == nvinfer1::PluginFormat::kNCHW));
  }
}

nvinfer1::Dims SlicePlugin::getOutputDimensions(int index,
                                                const nvinfer1::Dims *inputs,
                                                int nb_input_dims) {
  auto in_dims = inputs[0];
  nvinfer1::Dims out_dims = in_dims;
  for (size_t i = 0; i < axes_.size(); i++) {
    int start = starts_[i];
    int end = ends_[i];
    out_dims.d[axes_[i] - 1] = end - start;
  }
  return out_dims;
}

int SlicePlugin::enqueue(int batch_size, const void *const *inputs,
                         void **outputs, void *workspace, hipStream_t stream) {
  auto input_dims = getInputDims(0);

  // notice input dims is [C, H, W], add input batch dim here
  auto out_dims = getOutputDimensions(0, &input_dims, 1);
  input_dims.nbDims += 1;
  out_dims.nbDims += 1;
  for (auto i = input_dims.nbDims; i > 0; --i) {
    input_dims.d[i] = input_dims.d[i - 1];
    out_dims.d[i] = out_dims.d[i - 1];
  }
  input_dims.d[0] = batch_size;
  out_dims.d[0] = batch_size;

  auto num_dims = input_dims.nbDims;
  size_t out_num = ProductDim(out_dims);

  std::vector<int> seg_offsets;
  std::vector<int> offsets;
  std::vector<int> extends;

  offsets.resize(num_dims);
  extends.resize(num_dims);
  seg_offsets.resize(num_dims);

  seg_offsets[num_dims - 1] = 1;
  for (int i = num_dims - 2; i >= 0; i--) {
    seg_offsets[i] = input_dims.d[i + 1] * seg_offsets[i + 1];
  }
  for (size_t i = 0; i < num_dims; ++i) {
    offsets[i] = 0;
    extends[i] = out_dims.d[i];
  }
  for (size_t i = 0; i < axes_.size(); ++i) {
    offsets[axes_[i]] = starts_[i];
  }

  std::vector<int> offset_info;
  for (size_t i = 0; i < num_dims; ++i) {
    offset_info.push_back(offsets[i]);
    offset_info.push_back(extends[i]);
    offset_info.push_back(seg_offsets[i]);
  }

  if (offset_temp_data_ == nullptr) {
    hipMalloc(&offset_temp_data_, 3 * num_dims * sizeof(int));
  }

  hipMemcpyAsync(offset_temp_data_, offset_info.data(),
                  sizeof(int) * 3 * num_dims, hipMemcpyHostToDevice,
                  copy_stream_);

  hipEventRecord(copy_event_, copy_stream_);
  hipStreamWaitEvent(stream, copy_event_, 0);

  int threads = 256;
  int blocks = (out_num + threads - 1) / threads;
  auto input_type = getDataType();
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. Slice-->fp32";
    const float *input1 = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    SliceKernel<float><<<blocks, threads, 3 * num_dims * sizeof(int), stream>>>(
        out_num, num_dims, input1, offset_temp_data_, output);
  } else if (input_type == nvinfer1::DataType::kHALF) {
    VLOG(1) << "TRT Plugin DataType selected. Slice-->fp16";
    const half *input1 = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    SliceKernel<half><<<blocks, threads, 3 * num_dims * sizeof(int), stream>>>(
        out_num, num_dims, input1, offset_temp_data_, output);
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The Slice TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

size_t SlicePlugin::getSerializationSize() {
  return getBaseSerializationSize() + SerializedSize(getPluginType()) +
         SerializedSize(starts_) + SerializedSize(ends_) +
         SerializedSize(axes_);
}

void SlicePlugin::serialize(void *buffer) {
  SerializeValue(&buffer, getPluginType());
  serializeBase(buffer);
  SerializeValue(&buffer, starts_);
  SerializeValue(&buffer, ends_);
  SerializeValue(&buffer, axes_);
}

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)
SlicePluginDynamic::SlicePluginDynamic(std::vector<int> starts,
                                       std::vector<int> ends,
                                       std::vector<int> axes, bool with_fp16)
    : starts_(starts), ends_(ends), axes_(axes) {
  with_fp16_ = with_fp16;
  hipEventCreate(&copy_event_);
  hipStreamCreate(&copy_stream_);
}

SlicePluginDynamic::SlicePluginDynamic(void const *serialData,
                                       size_t serialLength) {
  DeserializeValue(&serialData, &serialLength, &starts_);
  DeserializeValue(&serialData, &serialLength, &ends_);
  DeserializeValue(&serialData, &serialLength, &axes_);
  DeserializeValue(&serialData, &serialLength, &with_fp16_);
  hipEventCreate(&copy_event_);
  hipStreamCreate(&copy_stream_);
}

void SlicePluginDynamic::destroy() {
  hipStreamDestroy(copy_stream_);
  hipEventDestroy(copy_event_);
  hipFree(offset_temp_data_);
  delete this;
}

int SlicePluginDynamic::initialize() { return 0; }

size_t SlicePluginDynamic::getSerializationSize() const {
  size_t size = SerializedSize(starts_) + SerializedSize(ends_) +
                SerializedSize(axes_) + SerializedSize(with_fp16_);

  return size;
}

void SlicePluginDynamic::serialize(void *buffer) const {
  SerializeValue(&buffer, starts_);
  SerializeValue(&buffer, ends_);
  SerializeValue(&buffer, axes_);
  SerializeValue(&buffer, with_fp16_);
}

nvinfer1::DimsExprs SlicePluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  auto in_dims = inputs[0];
  nvinfer1::DimsExprs ret = in_dims;
  // start, ends should greater 0
  for (size_t i = 0; i < axes_.size(); i++) {
    int start = starts_[i];
    int end = ends_[i];
    ret.d[axes_[i]] = expr_builder.constant(end - start);
  }
  return ret;
}

bool SlicePluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
      return (in.type == nvinfer1::DataType::kFLOAT ||
              in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];
  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType SlicePluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The Slice Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT ||
                     input_types[0] == nvinfer1::DataType::kHALF),
                    true, platform::errors::InvalidArgument(
                              "The input type should be half or float"));
  return input_types[0];
}

int SlicePluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                                const nvinfer1::PluginTensorDesc *output_desc,
                                const void *const *inputs, void *const *outputs,
                                void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  auto out_dims = output_desc[0].dims;
  auto num_dims = input_dims.nbDims;
  size_t out_num = ProductDim(out_dims);

  std::vector<int> seg_offsets;
  std::vector<int> offsets;
  std::vector<int> extends;

  offsets.resize(num_dims);
  extends.resize(num_dims);
  seg_offsets.resize(num_dims);

  seg_offsets[num_dims - 1] = 1;
  for (int i = num_dims - 2; i >= 0; i--) {
    seg_offsets[i] = input_dims.d[i + 1] * seg_offsets[i + 1];
  }

  for (size_t i = 0; i < num_dims; ++i) {
    offsets[i] = 0;
    extends[i] = out_dims.d[i];
  }
  for (size_t i = 0; i < axes_.size(); ++i) {
    offsets[axes_[i]] = starts_[i];
  }

  std::vector<int> offset_info;
  for (size_t i = 0; i < num_dims; ++i) {
    offset_info.push_back(offsets[i]);
    offset_info.push_back(extends[i]);
    offset_info.push_back(seg_offsets[i]);
  }

  if (offset_temp_data_ == nullptr) {
    hipMalloc(&offset_temp_data_, 3 * num_dims * sizeof(int));
  }

  hipMemcpyAsync(offset_temp_data_, offset_info.data(),
                  sizeof(int) * 3 * num_dims, hipMemcpyHostToDevice,
                  copy_stream_);

  hipEventRecord(copy_event_, copy_stream_);
  hipStreamWaitEvent(stream, copy_event_, 0);

  int threads = 256;
  int blocks = (out_num + threads - 1) / threads;
  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. Slice-->fp32";
    const float *input1 = static_cast<const float *>(inputs[0]);
    float *output = static_cast<float *>(outputs[0]);
    SliceKernel<float><<<blocks, threads, 3 * num_dims * sizeof(int), stream>>>(
        out_num, num_dims, input1, offset_temp_data_, output);
  } else if (input_type == nvinfer1::DataType::kHALF) {
    VLOG(1) << "TRT Plugin DataType selected. Slice-->fp16";
    const half *input1 = static_cast<const half *>(inputs[0]);
    half *output = static_cast<half *>(outputs[0]);
    SliceKernel<half><<<blocks, threads, 3 * num_dims * sizeof(int), stream>>>(
        out_num, num_dims, input1, offset_temp_data_, output);
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The Slice TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
