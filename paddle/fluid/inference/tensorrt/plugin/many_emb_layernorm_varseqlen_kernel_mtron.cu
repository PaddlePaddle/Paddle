#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
// SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
// AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <cassert>
#include <cstring>
#include <iostream>
#include <vector>

#include "NvInfer.h"
#include "paddle/fluid/inference/tensorrt/plugin/common/common.cuh"
#include "paddle/fluid/inference/tensorrt/plugin/many_emb_layernorm_varseqlen_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

template <typename T, unsigned TPB>
__global__ void embLayerNormKernelMTron_2(int32_t ld,
                                          int32_t const* inputIds0,
                                          int32_t const* inputIds1,
                                          float const* beta,
                                          float const* gamma,
                                          T const* mIdsEmbDev0,
                                          T const* mIdsEmbDev1,
                                          int32_t IdsSize0,
                                          int32_t IdsSize1,
                                          T* output,
                                          T* skip) {
  hipcub::Sum pairSum;
  int32_t const s = blockIdx.x;
  int32_t const b = blockIdx.y;
  int32_t const sumS = inputIds0[b];
  int32_t const s_b = inputIds0[b + 1] - sumS;
  if (s >= s_b) {
    return;  // This CTA has nothing to do
  }
  T const rld = T(1.f) / T(ld);
  const int32_t seqPos = sumS + s;
  extern __shared__ int32_t word_id[];

  if (threadIdx.x == 0) {
    if (static_cast<int32_t const*>(inputIds1)[seqPos] < 0 ||
        static_cast<int32_t const*>(inputIds1)[seqPos] >= IdsSize1) {
      printf(
          "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
          "table: ID < 0 or ID > max ");
      return;
    } else {
      word_id[0] = static_cast<int32_t const*>(inputIds1)[seqPos];
    }
  }
  __syncthreads();

  // 2. load pos/tok/word embeddings and add them toghether
  // offset into embeddings is given by wordId * hidden_size
  const int32_t poffset = blockIdx.x * ld;
  const int32_t outOffset = seqPos * ld;
  // the output offset is given by b * (S*hidden_size) + s * hidden_size
  kvp<T> threadData(0, 0);

  for (int32_t it = threadIdx.x; it < ld; it += TPB) {
    T p(mIdsEmbDev0[poffset + it]);  // pos id
    T val = p;
    const int32_t offset = word_id[0] * ld;
    val += mIdsEmbDev1[offset + it];
    output[outOffset + it] = val;
    skip[outOffset + it] = val;

    const T rldval = rld * val;
    threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
  }

  // 3. layer norm on the sum
  layerNorm<T, T, float, TPB>(threadData, ld, outOffset, beta, gamma, output);
}

template <typename T, unsigned TPB>
__global__ void embLayerNormKernelMTron_3(int32_t ld,
                                          int32_t const* inputIds0,
                                          int32_t const* inputIds1,
                                          int32_t const* inputIds2,
                                          float const* beta,
                                          float const* gamma,
                                          T const* mIdsEmbDev0,
                                          T const* mIdsEmbDev1,
                                          T const* mIdsEmbDev2,
                                          int32_t IdsSize0,
                                          int32_t IdsSize1,
                                          int32_t IdsSize2,
                                          T* output,
                                          T* skip) {
  hipcub::Sum pairSum;
  const int32_t s = blockIdx.x;
  const int32_t b = blockIdx.y;
  const int32_t sumS = inputIds0[b];
  const int32_t s_b = inputIds0[b + 1] - sumS;
  if (s >= s_b) {
    return;  // This CTA has nothing to do
  }
  const T rld = T(1.f) / T(ld);
  const int32_t seqPos = sumS + s;
  extern __shared__ int32_t word_id[];

  if (threadIdx.x == 0) {
    if (static_cast<int32_t const*>(inputIds1)[seqPos] < 0 ||
        static_cast<int32_t const*>(inputIds1)[seqPos] >= IdsSize1) {
      printf(
          "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
          "table: ID < 0 or ID > max ");
      return;
    } else {
      word_id[0] = static_cast<int32_t const*>(inputIds1)[seqPos];
    }

    if (static_cast<int32_t const*>(inputIds2)[seqPos] < 0 ||
        static_cast<int32_t const*>(inputIds2)[seqPos] >= IdsSize2) {
      printf(
          "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
          "table: ID < 0 or ID > max ");
      return;
    } else {
      word_id[1] = static_cast<int32_t const*>(inputIds2)[seqPos];
    }
  }
  __syncthreads();

  // 2. load pos/tok/word embeddings and add them toghether
  // offset into embeddings is given by wordId * hidden_size
  const int32_t poffset = blockIdx.x * ld;
  const int32_t outOffset = seqPos * ld;
  // the output offset is given by b * (S*hidden_size) + s * hidden_size
  kvp<T> threadData(0, 0);

  for (int32_t it = threadIdx.x; it < ld; it += TPB) {
    T p(mIdsEmbDev0[poffset + it]);  // pos id
    T val = p;
    const int32_t offset0 = word_id[0] * ld;
    val += mIdsEmbDev1[offset0 + it];
    const int32_t offset1 = word_id[1] * ld;
    val += mIdsEmbDev2[offset1 + it];
    output[outOffset + it] = val;
    skip[outOffset + it] = val;

    const T rldval = rld * val;
    threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
  }

  // 3. layer norm on the sum
  layerNorm<T, T, float, TPB>(threadData, ld, outOffset, beta, gamma, output);
}

template <typename T, unsigned TPB>
__global__ void embLayerNormKernelMTron_4(int32_t ld,
                                          int32_t const* inputIds0,
                                          int32_t const* inputIds1,
                                          int32_t const* inputIds2,
                                          int32_t const* inputIds3,
                                          float const* beta,
                                          float const* gamma,
                                          T const* mIdsEmbDev0,
                                          T const* mIdsEmbDev1,
                                          T const* mIdsEmbDev2,
                                          T const* mIdsEmbDev3,
                                          int32_t IdsSize0,
                                          int32_t IdsSize1,
                                          int32_t IdsSize2,
                                          int32_t IdsSize3,
                                          T* output,
                                          T* skip) {
  hipcub::Sum pairSum;
  const int32_t s = blockIdx.x;
  const int32_t b = blockIdx.y;
  const int32_t sumS = inputIds0[b];
  const int32_t s_b = inputIds0[b + 1] - sumS;
  if (s >= s_b) {
    return;  // This CTA has nothing to do
  }
  const T rld = T(1.f) / T(ld);
  const int32_t seqPos = sumS + s;
  extern __shared__ int32_t word_id[];

  if (threadIdx.x == 0) {
    if (static_cast<int32_t const*>(inputIds1)[seqPos] < 0 ||
        static_cast<int32_t const*>(inputIds1)[seqPos] >= IdsSize1) {
      printf(
          "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
          "table: ID < 0 or ID > max ");
      return;
    } else {
      word_id[0] = static_cast<int32_t const*>(inputIds1)[seqPos];
    }

    if (static_cast<int32_t const*>(inputIds2)[seqPos] < 0 ||
        static_cast<int32_t const*>(inputIds2)[seqPos] >= IdsSize2) {
      printf(
          "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
          "table: ID < 0 or ID > max ");
      return;
    } else {
      word_id[1] = static_cast<int32_t const*>(inputIds2)[seqPos];
    }

    if (static_cast<int32_t const*>(inputIds3)[seqPos] < 0 ||
        static_cast<int32_t const*>(inputIds3)[seqPos] >= IdsSize3) {
      printf(
          "Error!!!!!!(embLayerNormVarSeqlenPlugin): ID cannot be lookup "
          "table: ID < 0 or ID > max ");
      return;
    } else {
      word_id[2] = static_cast<int32_t const*>(inputIds3)[seqPos];
    }
  }
  __syncthreads();

  // 2. load pos/tok/word embeddings and add them toghether
  // offset into embeddings is given by wordId * hidden_size
  const int32_t poffset = blockIdx.x * ld;
  const int32_t outOffset = seqPos * ld;
  // the output offset is given by b * (S*hidden_size) + s * hidden_size
  kvp<T> threadData(0, 0);

  for (int32_t it = threadIdx.x; it < ld; it += TPB) {
    T p(mIdsEmbDev0[poffset + it]);  // pos id
    T val = p;
    const int32_t offset0 = word_id[0] * ld;
    val += mIdsEmbDev1[offset0 + it];
    const int32_t offset1 = word_id[1] * ld;
    val += mIdsEmbDev2[offset1 + it];
    const int32_t offset2 = word_id[2] * ld;
    val += mIdsEmbDev3[offset2 + it];
    output[outOffset + it] = val;
    skip[outOffset + it] = val;

    const T rldval = rld * val;
    threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
  }

  // 3. layer norm on the sum
  layerNorm<T, T, float, TPB>(threadData, ld, outOffset, beta, gamma, output);
}
template <typename T>
int32_t embSkipLayerNormMTron_2(hipStream_t stream,
                                int32_t ld,
                                int32_t B,
                                int32_t S,
                                int32_t const* inputIds0,
                                int32_t const* inputIds1,
                                int32_t nbLookupTables,
                                float const* beta,
                                float const* gamma,
                                T const* mIdsEmbDev0,
                                T const* mIdsEmbDev1,
                                int32_t IdsSize0,
                                int32_t IdsSize1,
                                T* output,
                                T* skip) {
  constexpr int32_t tpb = 256;
  dim3 const grid(S, B, 1);
  dim3 const block(tpb, 1, 1);
  size_t cache_size = sizeof(int32_t) * (nbLookupTables - 1);
  embLayerNormKernelMTron_2<T, tpb>
      <<<grid, block, cache_size, stream>>>(ld,
                                            inputIds0,
                                            inputIds1,
                                            beta,
                                            gamma,
                                            mIdsEmbDev0,
                                            mIdsEmbDev1,
                                            IdsSize0,
                                            IdsSize1,
                                            output,
                                            skip);
  return hipPeekAtLastError();
}

template <typename T>
int32_t embSkipLayerNormMTron_3(hipStream_t stream,
                                int32_t ld,
                                int32_t B,
                                int32_t S,
                                int32_t const* inputIds0,
                                int32_t const* inputIds1,
                                int32_t const* inputIds2,
                                int32_t nbLookupTables,
                                float const* beta,
                                float const* gamma,
                                T const* mIdsEmbDev0,
                                T const* mIdsEmbDev1,
                                T const* mIdsEmbDev2,
                                int32_t IdsSize0,
                                int32_t IdsSize1,
                                int32_t IdsSize2,
                                T* output,
                                T* skip) {
  constexpr int32_t tpb = 256;
  dim3 const grid(S, B, 1);
  dim3 const block(tpb, 1, 1);
  size_t cache_size = sizeof(int32_t) * (nbLookupTables - 1);
  embLayerNormKernelMTron_3<T, tpb>
      <<<grid, block, cache_size, stream>>>(ld,
                                            inputIds0,
                                            inputIds1,
                                            inputIds2,
                                            beta,
                                            gamma,
                                            mIdsEmbDev0,
                                            mIdsEmbDev1,
                                            mIdsEmbDev2,
                                            IdsSize0,
                                            IdsSize1,
                                            IdsSize2,
                                            output,
                                            skip);
  return hipPeekAtLastError();
}

template <typename T>
int32_t embSkipLayerNormMTron_4(hipStream_t stream,
                                int32_t ld,
                                int32_t B,
                                int32_t S,
                                int32_t const* inputIds0,
                                int32_t const* inputIds1,
                                int32_t const* inputIds2,
                                int32_t const* inputIds3,
                                int32_t nbLookupTables,
                                float const* beta,
                                float const* gamma,
                                T const* mIdsEmbDev0,
                                T const* mIdsEmbDev1,
                                T const* mIdsEmbDev2,
                                T const* mIdsEmbDev3,
                                int32_t IdsSize0,
                                int32_t IdsSize1,
                                int32_t IdsSize2,
                                int32_t IdsSize3,
                                T* output,
                                T* skip) {
  constexpr int32_t tpb = 256;
  dim3 const grid(S, B, 1);
  dim3 const block(tpb, 1, 1);
  size_t cache_size = sizeof(int32_t) * (nbLookupTables - 1);
  embLayerNormKernelMTron_4<T, tpb>
      <<<grid, block, cache_size, stream>>>(ld,
                                            inputIds0,
                                            inputIds1,
                                            inputIds2,
                                            inputIds3,
                                            beta,
                                            gamma,
                                            mIdsEmbDev0,
                                            mIdsEmbDev1,
                                            mIdsEmbDev2,
                                            mIdsEmbDev3,
                                            IdsSize0,
                                            IdsSize1,
                                            IdsSize2,
                                            IdsSize3,
                                            output,
                                            skip);
  return hipPeekAtLastError();
}

template int32_t embSkipLayerNormMTron_2<float>(hipStream_t,
                                                int32_t,
                                                int32_t,
                                                int32_t,
                                                int32_t const*,
                                                int32_t const*,
                                                int32_t,
                                                float const*,
                                                float const*,
                                                float const*,
                                                float const*,
                                                int32_t,
                                                int32_t,
                                                float*,
                                                float*);

template int32_t embSkipLayerNormMTron_3<float>(hipStream_t,
                                                int32_t,
                                                int32_t,
                                                int32_t,
                                                int32_t const*,
                                                int32_t const*,
                                                int32_t const*,
                                                int32_t,
                                                float const*,
                                                float const*,
                                                float const*,
                                                float const*,
                                                float const*,
                                                int32_t,
                                                int32_t,
                                                int32_t,
                                                float*,
                                                float*);

template int32_t embSkipLayerNormMTron_4<float>(hipStream_t,
                                                int32_t,
                                                int32_t,
                                                int32_t,
                                                int32_t const*,
                                                int32_t const*,
                                                int32_t const*,
                                                int32_t const*,
                                                int32_t,
                                                float const*,
                                                float const*,
                                                float const*,
                                                float const*,
                                                float const*,
                                                float const*,
                                                int32_t,
                                                int32_t,
                                                int32_t,
                                                int32_t,
                                                float*,
                                                float*);

template int32_t embSkipLayerNormMTron_2<half>(hipStream_t,
                                               int32_t,
                                               int32_t,
                                               int32_t,
                                               int32_t const*,
                                               int32_t const*,
                                               int32_t,
                                               float const*,
                                               float const*,
                                               half const*,
                                               half const*,
                                               int32_t,
                                               int32_t,
                                               half*,
                                               half*);

template int32_t embSkipLayerNormMTron_3<half>(hipStream_t,
                                               int32_t,
                                               int32_t,
                                               int32_t,
                                               int32_t const*,
                                               int32_t const*,
                                               int32_t const*,
                                               int32_t,
                                               float const*,
                                               float const*,
                                               half const*,
                                               half const*,
                                               half const*,
                                               int32_t,
                                               int32_t,
                                               int32_t,
                                               half*,
                                               half*);

template int32_t embSkipLayerNormMTron_4<half>(hipStream_t,
                                               int32_t,
                                               int32_t,
                                               int32_t,
                                               int32_t const*,
                                               int32_t const*,
                                               int32_t const*,
                                               int32_t const*,
                                               int32_t,
                                               float const*,
                                               float const*,
                                               half const*,
                                               half const*,
                                               half const*,
                                               half const*,
                                               int32_t,
                                               int32_t,
                                               int32_t,
                                               int32_t,
                                               half*,
                                               half*);

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
