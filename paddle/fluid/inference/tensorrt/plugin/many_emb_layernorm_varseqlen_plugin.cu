#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
// SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
// AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/many_emb_layernorm_varseqlen_plugin.h"
#include <hip/hip_runtime.h>
#include <cstring>
#include <vector>
#include "NvInfer.h"
#include "common/serialize.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

constexpr size_t threadsPerCta128 = 2 * 2 * 32;
constexpr size_t threadsPerCta256 = 1 * 4 * 32;
constexpr size_t threadsPerCta384 = 1 * 8 * 32;
// The number of xmmas in the M dimension. We use one uint32_t per XMMA in the M
// dimension: (s + 16*warps_m - 1) / (16*warps_m);
constexpr size_t xmmasM128 = 4;
constexpr size_t xmmasM256 = 16;
constexpr size_t xmmasM384 = 24;
// Packed mask size per batch. Layout is XMMAS_M * THREADS_PER_CTA.
constexpr size_t packedMaskSize128 = xmmasM128 * threadsPerCta128;
constexpr size_t packedMaskSize256 = xmmasM256 * threadsPerCta256;
constexpr size_t packedMaskSize384 = xmmasM384 * threadsPerCta384;
char const* EMB_LAYER_NORM_VAR_SEQLEN_VERSION_HFACE{"2"};
char const* EMB_LAYER_NORM_VAR_SEQLEN_VERSION_MTRON{"3"};
char const* EMB_LAYER_NORM_VAR_SEQLEN_NAME{"ManyEmbLayerNormPluginDynamic"};
// Static class fields initialization
nvinfer1::PluginFieldCollection EmbLayerNormVarSeqlenPluginBaseCreator::mFC{};
std::vector<nvinfer1::PluginField>
    EmbLayerNormVarSeqlenPluginBaseCreator::mPluginAttributes;

EmbLayerNormVarSeqlenPluginBase::EmbLayerNormVarSeqlenPluginBase(
    std::string const& name,
    nvinfer1::DataType const type,
    nvinfer1::Weights const& beta,
    nvinfer1::Weights const& gamma,
    const std::vector<nvinfer1::Weights>& IdsEmb)
    : mLayerName(name),
      mLd(beta.count),
      mType(type),
      mIdsEmb_(IdsEmb),
      nbLookupTables_(static_cast<int>(IdsEmb.size())) {
  // Assuming Weights.count is the number of elements and not bytes
  assert(beta.count == gamma.count);
  mBeta.convertAndCopy(beta, nvinfer1::DataType::kFLOAT);
  mGamma.convertAndCopy(gamma, nvinfer1::DataType::kFLOAT);
  copyToDevice(mGamma, sizeof(float) * mGamma.count, mGammaDev);
  copyToDevice(mBeta, sizeof(float) * mBeta.count, mBetaDev);
  for (size_t i = 0; i < mIdsEmb_.size(); ++i) {
    assert(mIdsEmb_[i].count % mLd == 0);
    mIdsVocabSize.push_back(int32_t(mIdsEmb_[i].count / mLd));
    WeightsWithOwnership tem_weight;
    tem_weight.convertAndCopy(mIdsEmb_[i], mType);
    void* cudaMem{nullptr};
    PADDLE_ENFORCE_GPU_SUCCESS(
        hipMalloc(&cudaMem, getWeightsSize(tem_weight, mType)));
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(cudaMem,
                                          tem_weight.values,
                                          getWeightsSize(tem_weight, mType),
                                          hipMemcpyHostToDevice));
    mIdsEmbDev.push_back(cudaMem);
  }
}

EmbLayerNormVarSeqlenPluginBase::EmbLayerNormVarSeqlenPluginBase(
    std::string const& name, void const* data, size_t length)
    : mLayerName(name),
      mGammaDev(nullptr),
      mBetaDev(nullptr),
      mIdsEmbDev{},
      mIdsEmb_{} {
  // Deserialize in the same order as serialization
  deserialize_value(&data, &length, &mType);
  deserialize_value(&data, &length, &mLd);
  deserialize_value(&data, &length, &nbLookupTables_);
  for (int32_t i = 0; i < nbLookupTables_; ++i) {
    int32_t tem;
    deserialize_value(&data, &length, &tem);
    mIdsVocabSize.push_back(tem);
  }
  char const* d = static_cast<char const*>(data);
  mBeta.convertAndCopy(d, mLd, nvinfer1::DataType::kFLOAT);
  mGamma.convertAndCopy(d, mLd, nvinfer1::DataType::kFLOAT);
  for (int32_t i = 0; i < nbLookupTables_; ++i) {
    nvinfer1::Weights pre_tem_weight;
    pre_tem_weight.type = mType;
    pre_tem_weight.count = mLd * size_t(mIdsVocabSize[i]);
    const auto nbBytes = mLd * size_t(mIdsVocabSize[i]) * getElementSize(mType);
    auto destBuf = new char[nbBytes];
    pre_tem_weight.values = destBuf;
    std::copy_n(d, nbBytes, destBuf);
    d += nbBytes;
    mIdsEmb_.push_back(pre_tem_weight);
  }
}

EmbLayerNormVarSeqlenPluginHFace::EmbLayerNormVarSeqlenPluginHFace(
    std::string const& name,
    nvinfer1::DataType const type,
    nvinfer1::Weights const& beta,
    nvinfer1::Weights const& gamma,
    const std::vector<nvinfer1::Weights>& IdsEmb)
    : EmbLayerNormVarSeqlenPluginBase(name, type, beta, gamma, IdsEmb) {}

EmbLayerNormVarSeqlenPluginHFace::EmbLayerNormVarSeqlenPluginHFace(
    std::string const& name, void const* data, size_t length)
    : EmbLayerNormVarSeqlenPluginBase(name, data, length) {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginHFace deserialize");
}

EmbLayerNormVarSeqlenPluginMTron::EmbLayerNormVarSeqlenPluginMTron(
    std::string const& name,
    nvinfer1::DataType const type,
    nvinfer1::Weights const& beta,
    nvinfer1::Weights const& gamma,
    const std::vector<nvinfer1::Weights>& IdsEmb)
    : EmbLayerNormVarSeqlenPluginBase(name, type, beta, gamma, IdsEmb) {}

EmbLayerNormVarSeqlenPluginMTron::EmbLayerNormVarSeqlenPluginMTron(
    std::string const& name, void const* data, size_t length)
    : EmbLayerNormVarSeqlenPluginBase(name, data, length) {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron deserialize");
}

// IPluginV2DynamicExt Methods
nvinfer1::IPluginV2DynamicExt* EmbLayerNormVarSeqlenPluginHFace::clone()
    const noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron clone");
  auto p = new EmbLayerNormVarSeqlenPluginMTron(
      mLayerName, mType, mBeta, mGamma, mIdsEmb_);
  p->setPluginNamespace(mNamespace.c_str());
  return p;
}

nvinfer1::IPluginV2DynamicExt* EmbLayerNormVarSeqlenPluginMTron::clone()
    const noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron clone");
  auto p = new EmbLayerNormVarSeqlenPluginMTron(
      mLayerName, mType, mBeta, mGamma, mIdsEmb_);
  p->setPluginNamespace(mNamespace.c_str());
  return p;
}

nvinfer1::DimsExprs EmbLayerNormVarSeqlenPluginHFace::getOutputDimensions(
    int32_t outputIndex,
    nvinfer1::DimsExprs const* inputs,
    int32_t nbInputs,
    nvinfer1::IExprBuilder& exprBuilder) noexcept {
  for (int i = 1; i < nbInputs - 1; ++i) {
    assert(inputs[i].nbDims == 1);                 // seq length
    assert(inputs[i].nbDims == inputs[1].nbDims);  // same shape
  }
  assert(inputs[0].nbDims == 1);  // pos_id: B+1
  assert(outputIndex == 0 || outputIndex == 1);
  if (outputIndex == 0) {
    nvinfer1::DimsExprs ret;
    ret.nbDims = 4;
    ret.d[0] = inputs[1].d[0];  // sum of seq length
    ret.d[1] = exprBuilder.constant(mLd);
    ret.d[2] = exprBuilder.constant(1);
    ret.d[3] = exprBuilder.constant(1);
    return ret;
  }

  // This is a hack: we just report some mask size and rely the plugins to play
  // nicely together.
  //      At runtime, depending on the actual maxSeqlen, the size might be
  //      different.
  int32_t maskSize_ = packedMaskSize384;
  auto maskSize = exprBuilder.constant(maskSize_);
  auto fp16maskSize = exprBuilder.operation(
      nvinfer1::DimensionOperation::kPROD, *maskSize, *exprBuilder.constant(2));
  auto Bplus1 = inputs[0].d[0];  // pos_id
  auto one = exprBuilder.constant(1);
  auto B =
      exprBuilder.operation(nvinfer1::DimensionOperation::kSUB, *Bplus1, *one);
  nvinfer1::DimsExprs ret;
  ret.nbDims = 2;
  ret.d[0] = B;
  ret.d[1] = fp16maskSize;
  return ret;
}

nvinfer1::DimsExprs EmbLayerNormVarSeqlenPluginMTron::getOutputDimensions(
    int32_t outputIndex,
    nvinfer1::DimsExprs const* inputs,
    int32_t nbInputs,
    nvinfer1::IExprBuilder& exprBuilder) noexcept {
  // Input should be input ids and token ids and cumulative seqlens
  // Output should be the embeddings tensor and mask indices
  for (int i = 1; i < nbInputs - 1; ++i) {
    assert(inputs[i].nbDims == 1);                 // seq length
    assert(inputs[i].nbDims == inputs[1].nbDims);  // same shape
  }
  assert(inputs[0].nbDims == 1);  // pos_id: B+1
  assert(outputIndex == 0 || outputIndex == 1);
  nvinfer1::DimsExprs ret;
  ret.nbDims = 4;
  ret.d[0] = inputs[1].d[0];
  ret.d[1] = exprBuilder.constant(mLd);
  ret.d[2] = exprBuilder.constant(1);
  ret.d[3] = exprBuilder.constant(1);
  return ret;
}

bool EmbLayerNormVarSeqlenPluginBase::supportsFormatCombination(
    int32_t pos,
    nvinfer1::PluginTensorDesc const* inOut,
    int32_t nbInputs,
    int32_t nbOutputs) noexcept {
  assert(nbOutputs == 2);
  nvinfer1::PluginTensorDesc const& desc = inOut[pos];
  if (desc.format != nvinfer1::TensorFormat::kLINEAR) {
    return false;
  }
  if (pos == 0) {  // pos_id
    return desc.dims.nbDims == 1 && desc.type == nvinfer1::DataType::kINT32;
  }
  if (pos == 1) {  //  input_id
    return desc.dims.nbDims == 1 && desc.type == nvinfer1::DataType::kINT32;
  }
  nvinfer1::PluginTensorDesc const& prev = inOut[1];  // input_ids
  if (1 < pos &&
      pos < (nbInputs - 1)) {  // other ids: check it's the same as input_ids
    return desc.type == prev.type && desc.dims.nbDims == 1 &&
           desc.dims.d[0] == prev.dims.d[0];
  }
  if (pos == nbInputs - 1) {  // max seq length
    return desc.dims.nbDims == 1;
  }
  // embedded sequence
  if (pos == nbInputs) {
    return desc.type == mType && desc.dims.nbDims == 4 &&
           desc.dims.d[0] == inOut[1].dims.d[0] && desc.dims.d[2] == 1 &&
           desc.dims.d[3] == 1;
  }
  // mask
  return desc.type == nvinfer1::DataType::kHALF;
}

void checkConfigurationInputs(nvinfer1::DynamicPluginTensorDesc const* inputs,
                              int32_t nbInputs,
                              nvinfer1::DynamicPluginTensorDesc const* outputs,
                              int32_t nbOutputs) noexcept {
  // Validate input arguments
  // assert(nbInputs == 4);
  assert(nbOutputs == 2);
  assert(inputs[0].desc.dims.nbDims == 1);
  assert(inputs[0].desc.type == nvinfer1::DataType::kINT32);
  for (int i = 1; i < nbInputs - 1; ++i) {
    assert(inputs[i].desc.dims.nbDims == 1);
    assert(inputs[i].desc.dims.d[0] == inputs[1].desc.dims.d[0]);
    assert(inputs[i].desc.type == nvinfer1::DataType::kINT32);
  }
  assert(outputs[0].desc.dims.nbDims == 4);
  assert(static_cast<size_t>(outputs[0].desc.dims.d[0]) ==
         static_cast<size_t>(inputs[1].desc.dims.d[0]));
  assert(outputs[0].desc.dims.d[2] == 1);
  assert(outputs[0].desc.dims.d[3] == 1);
}

void EmbLayerNormVarSeqlenPluginHFace::configurePlugin(
    nvinfer1::DynamicPluginTensorDesc const* inputs,
    int32_t nbInputs,
    nvinfer1::DynamicPluginTensorDesc const* outputs,
    int32_t nbOutputs) noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginHFace configurePlugin");
  checkConfigurationInputs(inputs, nbInputs, outputs, nbOutputs);
  assert(static_cast<size_t>(outputs[0].desc.dims.d[1]) ==
         static_cast<size_t>(mLd));
  int32_t const B = inputs[0].desc.dims.d[0] - 1;
  // check mask
  assert(outputs[1].desc.dims.nbDims == 2);
  if (B > 0) {
    assert(outputs[1].desc.dims.d[0] == B);
  }
  assert((outputs[1].desc.dims.d[1] == 2 * packedMaskSize384) ||
         (outputs[1].desc.dims.d[1] == 2 * packedMaskSize128) ||
         (outputs[1].desc.dims.d[1] == 2 * packedMaskSize256));
  assert(outputs[0].desc.type == mType);
  assert(outputs[1].desc.type == nvinfer1::DataType::kHALF);
}

void EmbLayerNormVarSeqlenPluginMTron::configurePlugin(
    nvinfer1::DynamicPluginTensorDesc const* inputs,
    int32_t nbInputs,
    nvinfer1::DynamicPluginTensorDesc const* outputs,
    int32_t nbOutputs) noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron configurePlugin");
  checkConfigurationInputs(inputs, nbInputs, outputs, nbOutputs);
  assert(static_cast<size_t>(outputs[0].desc.dims.d[1]) ==
         static_cast<size_t>(mLd));
  assert(outputs[1].desc.dims.nbDims == 4);
  assert(static_cast<size_t>(outputs[1].desc.dims.d[0]) ==
         static_cast<size_t>(inputs[1].desc.dims.d[0]));
  assert(static_cast<size_t>(outputs[1].desc.dims.d[1]) ==
         static_cast<size_t>(mLd));
  assert(outputs[1].desc.dims.d[2] == 1);
  assert(outputs[1].desc.dims.d[3] == 1);

  assert(outputs[0].desc.type == mType);
  assert(outputs[1].desc.type == mType);
}

size_t EmbLayerNormVarSeqlenPluginBase::getWorkspaceSize(
    nvinfer1::PluginTensorDesc const* inputs,
    int32_t nbInputs,
    nvinfer1::PluginTensorDesc const* outputs,
    int32_t nbOutputs) const noexcept {
  return 0;
}

int32_t EmbLayerNormVarSeqlenPluginHFace::enqueue(
    nvinfer1::PluginTensorDesc const* inputDesc,
    nvinfer1::PluginTensorDesc const* outputDesc,
    void const* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) noexcept {
  int32_t const batchSize = inputDesc[0].dims.d[0] - 1;
  // read out the maximum sequence length from the dummy input
  int32_t const maxSeqlen = inputDesc[nbLookupTables_].dims.d[0];
  int32_t S = 384;
  if (maxSeqlen <= 128) {
    S = 128;
  } else if (maxSeqlen <= 192) {
    S = 192;
  } else if (maxSeqlen <= 256) {
    S = 256;
  }
  const float* beta = mBetaDev.get();
  const float* gamma = mGammaDev.get();
  int32_t** tem_inputs_ptr_dev;
  hipMalloc(reinterpret_cast<void**>(&tem_inputs_ptr_dev), sizeof(void*) * nbLookupTables_);
  hipMemcpy(tem_inputs_ptr_dev,
             inputs,
             sizeof(void*) * nbLookupTables_,
             hipMemcpyHostToDevice);
  int32_t* mIdsVocabSize_dev;
  hipMalloc(reinterpret_cast<void**>(&mIdsVocabSize_dev),
             sizeof(int32_t) * mIdsVocabSize.size());
  hipMemcpy(mIdsVocabSize_dev,
             &(mIdsVocabSize[0]),
             sizeof(int32_t) * mIdsVocabSize.size(),
             hipMemcpyHostToDevice);
  if (mType == nvinfer1::DataType::kFLOAT) {
    auto output = static_cast<float*>(outputs[0]);
    float** mIdsEmbDev_float;
    hipMalloc(reinterpret_cast<void**>(&mIdsEmbDev_float), sizeof(void*) * nbLookupTables_);
    hipMemcpy(mIdsEmbDev_float,
               &(mIdsEmbDev[0]),
               sizeof(void*) * nbLookupTables_,
               hipMemcpyHostToDevice);
    return embSkipLayerNormHFace<float>(stream,
                                        static_cast<int32_t>(mLd),
                                        batchSize,
                                        S,
                                        tem_inputs_ptr_dev,
                                        nbLookupTables_,
                                        beta,
                                        gamma,
                                        mIdsEmbDev_float,
                                        mIdsVocabSize_dev,
                                        output);
  } else if (mType == nvinfer1::DataType::kHALF) {
    auto output = static_cast<half*>(outputs[0]);
    half** mIdsEmbDev_half;
    hipMalloc(reinterpret_cast<void**>(&mIdsEmbDev_half), sizeof(void*) * nbLookupTables_);
    hipMemcpy(mIdsEmbDev_half,
               &(mIdsEmbDev[0]),
               sizeof(void*) * nbLookupTables_,
               hipMemcpyHostToDevice);
    return embSkipLayerNormHFace<half>(stream,
                                       static_cast<int32_t>(mLd),
                                       batchSize,
                                       S,
                                       tem_inputs_ptr_dev,
                                       nbLookupTables_,
                                       beta,
                                       gamma,
                                       mIdsEmbDev_half,
                                       mIdsVocabSize_dev,
                                       output);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "Unsupported type error, expected [kHALF,kFLOAT]"));
  }
  return STATUS_SUCCESS;
}

int32_t EmbLayerNormVarSeqlenPluginMTron::enqueue(
    nvinfer1::PluginTensorDesc const* inputDesc,
    nvinfer1::PluginTensorDesc const* outputDesc,
    void const* const* inputs,
    void* const* outputs,
    void* workspace,
    hipStream_t stream) noexcept {
  int32_t const batchSize = inputDesc[0].dims.d[0] - 1;
  // read out the maximum sequence length from the dummy input
  int32_t const maxSeqlen = inputDesc[nbLookupTables_].dims.d[0];
  int32_t S = 384;
  if (maxSeqlen <= 128) {
    S = 128;
  } else if (maxSeqlen <= 192) {
    S = 192;
  } else if (maxSeqlen <= 256) {
    S = 256;
  }
  const float* beta = mBetaDev.get();
  const float* gamma = mGammaDev.get();
  int32_t** tem_inputs_ptr_dev;
  hipMalloc(reinterpret_cast<void**>(&tem_inputs_ptr_dev), sizeof(void*) * nbLookupTables_);
  hipMemcpy(tem_inputs_ptr_dev,
             inputs,
             sizeof(void*) * nbLookupTables_,
             hipMemcpyHostToDevice);
  int32_t* mIdsVocabSize_dev;
  hipMalloc(reinterpret_cast<void**>(&mIdsVocabSize_dev),
             sizeof(int32_t) * mIdsVocabSize.size());
  hipMemcpy(mIdsVocabSize_dev,
             &(mIdsVocabSize[0]),
             sizeof(int32_t) * mIdsVocabSize.size(),
             hipMemcpyHostToDevice);
  if (mType == nvinfer1::DataType::kFLOAT) {
    auto output = static_cast<float*>(outputs[0]);
    auto skip = static_cast<float*>(outputs[1]);
    float** mIdsEmbDev_float;
    hipMalloc(reinterpret_cast<void**>(&mIdsEmbDev_float), sizeof(void*) * nbLookupTables_);
    hipMemcpy(mIdsEmbDev_float,
               &(mIdsEmbDev[0]),
               sizeof(void*) * nbLookupTables_,
               hipMemcpyHostToDevice);
    return embSkipLayerNormMTron<float>(stream,
                                        static_cast<int32_t>(mLd),
                                        batchSize,
                                        S,
                                        tem_inputs_ptr_dev,
                                        nbLookupTables_,
                                        beta,
                                        gamma,
                                        mIdsEmbDev_float,
                                        mIdsVocabSize_dev,
                                        output,
                                        skip);
  } else if (mType == nvinfer1::DataType::kHALF) {
    auto output = static_cast<half*>(outputs[0]);
    auto skip = static_cast<half*>(outputs[1]);
    half** mIdsEmbDev_half;
    hipMalloc(reinterpret_cast<void**>(&mIdsEmbDev_half), sizeof(void*) * nbLookupTables_);
    hipMemcpy(mIdsEmbDev_half,
               &(mIdsEmbDev[0]),
               sizeof(void*) * nbLookupTables_,
               hipMemcpyHostToDevice);
    return embSkipLayerNormMTron<half>(stream,
                                       static_cast<int32_t>(mLd),
                                       batchSize,
                                       S,
                                       tem_inputs_ptr_dev,
                                       nbLookupTables_,
                                       beta,
                                       gamma,
                                       mIdsEmbDev_half,
                                       mIdsVocabSize_dev,
                                       output,
                                       skip);
  } else {
    PADDLE_THROW(platform::errors::InvalidArgument(
        "Unsupported type error, expected [kHALF,kFLOAT]"));
  }
  return STATUS_SUCCESS;
}

// IPluginV2Ext Methods
nvinfer1::DataType EmbLayerNormVarSeqlenPluginBase::getOutputDataType(
    int32_t index,
    nvinfer1::DataType const* inputTypes,
    int32_t nbInputs) const noexcept {
  assert(index == 0 || index == 1);
  if (index == 0) {
    assert(mType == nvinfer1::DataType::kHALF ||
           mType == nvinfer1::DataType::kFLOAT);
    return mType;
  }
  return nvinfer1::DataType::kHALF;
}

// IPluginV2 Methods
char const* EmbLayerNormVarSeqlenPluginBase::getPluginType() const noexcept {
  return EMB_LAYER_NORM_VAR_SEQLEN_NAME;
}

char const* EmbLayerNormVarSeqlenPluginHFace::getPluginVersion()
    const noexcept {
  return EMB_LAYER_NORM_VAR_SEQLEN_VERSION_HFACE;
}

char const* EmbLayerNormVarSeqlenPluginMTron::getPluginVersion()
    const noexcept {
  return EMB_LAYER_NORM_VAR_SEQLEN_VERSION_MTRON;
}

int32_t EmbLayerNormVarSeqlenPluginBase::getNbOutputs() const noexcept {
  return 2;
}

int32_t EmbLayerNormVarSeqlenPluginHFace::initialize() noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginHFace initialize");
  return 0;
}

int32_t EmbLayerNormVarSeqlenPluginMTron::initialize() noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron initialize");
  return 0;
}

void EmbLayerNormVarSeqlenPluginHFace::terminate() noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginHFace terminate");
}

void EmbLayerNormVarSeqlenPluginMTron::terminate() noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron terminate");
}

size_t EmbLayerNormVarSeqlenPluginBase::getSerializationSize() const noexcept {
  size_t const wordSize = getElementSize(mType);
  return 2 * sizeof(float) * mLd                            // beta + gamma
         + sizeof(mType)                                    //
         + sizeof(mLd)                                      //
         + mIdsVocabSize.size() * sizeof(mIdsVocabSize[0])  //
         + wordSize * mLd *
               accumulate(
                   mIdsVocabSize.begin(), mIdsVocabSize.end(), 0)  // ids emb
         + sizeof(nbLookupTables_);  // numbers of lookup_table
}

void EmbLayerNormVarSeqlenPluginBase::serialize(void* buffer) const noexcept {
  serialize_value(&buffer, mType);
  serialize_value(&buffer, mLd);
  serialize_value(&buffer, nbLookupTables_);
  for (size_t i = 0; i < mIdsVocabSize.size(); ++i) {
    serialize_value(&buffer, mIdsVocabSize[i]);
  }
  char* d = static_cast<char*>(buffer);
  size_t const wordSize = getElementSize(mType);
  serFromDev(d, mBetaDev.get(), mLd);
  serFromDev(d, mGammaDev.get(), mLd);
  for (size_t i = 0; i < mIdsEmbDev.size(); ++i) {
    serFromDev(d,
               static_cast<char*>(mIdsEmbDev[i]),
               mLd * mIdsVocabSize[i] * wordSize);
  }
}

void EmbLayerNormVarSeqlenPluginBase::destroy() noexcept {
  // This gets called when the network containing plugin is destroyed
  mBetaDev.reset(nullptr);
  mGammaDev.reset(nullptr);
  for (size_t i = 0; i < mIdsEmbDev.size(); ++i) {
    hipFree(mIdsEmbDev[i]);
  }
  delete this;
}

void EmbLayerNormVarSeqlenPluginHFace::destroy() noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginHFace destroy");
  EmbLayerNormVarSeqlenPluginBase::destroy();
}

void EmbLayerNormVarSeqlenPluginMTron::destroy() noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenPluginMTron destroy");
  EmbLayerNormVarSeqlenPluginBase::destroy();
}

void EmbLayerNormVarSeqlenPluginBase::setPluginNamespace(
    char const* libNamespace) noexcept {
  mNamespace = libNamespace;
}

char const* EmbLayerNormVarSeqlenPluginBase::getPluginNamespace()
    const noexcept {
  return mNamespace.c_str();
}

EmbLayerNormVarSeqlenPluginBaseCreator::
    EmbLayerNormVarSeqlenPluginBaseCreator() {}

char const* EmbLayerNormVarSeqlenPluginBaseCreator::getPluginName()
    const noexcept {
  return EMB_LAYER_NORM_VAR_SEQLEN_NAME;
}

char const* EmbLayerNormVarSeqlenPluginHFaceCreator::getPluginVersion()
    const noexcept {
  return EMB_LAYER_NORM_VAR_SEQLEN_VERSION_HFACE;
}

char const* EmbLayerNormVarSeqlenPluginMTronCreator::getPluginVersion()
    const noexcept {
  return EMB_LAYER_NORM_VAR_SEQLEN_VERSION_MTRON;
}

nvinfer1::PluginFieldCollection const*
EmbLayerNormVarSeqlenPluginBaseCreator::getFieldNames() noexcept {
  return &mFC;
}

bool initializeFields(nvinfer1::PluginFieldCollection const* fc,
                      nvinfer1::Weights& beta,
                      nvinfer1::Weights& gamma,
                      std::vector<nvinfer1::Weights>& IdsEmb) {
  bool output_fp16 = false;
  for (int32_t i = 0; i < fc->nbFields; i++) {
    std::string field_name(fc->fields[i].name);
    if (field_name.compare("bert_embeddings_layernorm_beta") == 0) {
      TRANSFORMER_DEBUG_MSG("Building bert_embeddings_layernorm_beta...");
      beta.values = fc->fields[i].data;
      beta.count = fc->fields[i].length;
      beta.type = fieldTypeToDataType(fc->fields[i].type);
    }

    if (field_name.compare("bert_embeddings_layernorm_gamma") == 0) {
      TRANSFORMER_DEBUG_MSG("Building bert_embeddings_layernorm_gamma...");
      gamma.values = fc->fields[i].data;
      gamma.count = fc->fields[i].length;
      gamma.type = fieldTypeToDataType(fc->fields[i].type);
    }

    if (field_name.compare("output_fp16") == 0) {
      TRANSFORMER_DEBUG_MSG("Building output_fp16...");
      assert(fc->fields[i].type == nvinfer1::PluginFieldType::kINT32);
      output_fp16 = static_cast<int32_t const*>(fc->fields[i].data)[0] != 0;
    }
    if (field_name.compare("bert_embeddings_word_embeddings_" +
                           std::to_string(i - 3)) == 0) {
      TRANSFORMER_DEBUG_MSG(
          ("bert_embeddings_word_embeddings_" + std::to_string(i - 3)).c_str());
      nvinfer1::Weights tem;
      tem.values = fc->fields[i].data;
      tem.count = fc->fields[i].length;
      tem.type = fieldTypeToDataType(fc->fields[i].type);
      IdsEmb.push_back(tem);
    }
  }
  return output_fp16;
}

nvinfer1::IPluginV2* EmbLayerNormVarSeqlenPluginHFaceCreator::createPlugin(
    char const* name, nvinfer1::PluginFieldCollection const* fc) noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenHFace createPlugin");
  nvinfer1::Weights beta;
  nvinfer1::Weights gamma;
  std::vector<nvinfer1::Weights> IdsEmb;
  bool output_fp16 = initializeFields(fc, beta, gamma, IdsEmb);
  TRANSFORMER_DEBUG_MSG("Building the Plugin...");
  EmbLayerNormVarSeqlenPluginHFace* p = new EmbLayerNormVarSeqlenPluginHFace(
      name,
      output_fp16 ? nvinfer1::DataType::kHALF : nvinfer1::DataType::kFLOAT,
      beta,
      gamma,
      IdsEmb);

  return p;
}

nvinfer1::IPluginV2* EmbLayerNormVarSeqlenPluginMTronCreator::createPlugin(
    char const* name, nvinfer1::PluginFieldCollection const* fc) noexcept {
  TRANSFORMER_DEBUG_MSG("EmbLayerNormVarSeqlenMTron createPlugin");
  nvinfer1::Weights beta;
  nvinfer1::Weights gamma;
  std::vector<nvinfer1::Weights> IdsEmb;
  bool output_fp16 = initializeFields(fc, beta, gamma, IdsEmb);
  TRANSFORMER_DEBUG_MSG("Building the Plugin...");
  EmbLayerNormVarSeqlenPluginMTron* p = new EmbLayerNormVarSeqlenPluginMTron(
      name,
      output_fp16 ? nvinfer1::DataType::kHALF : nvinfer1::DataType::kFLOAT,
      beta,
      gamma,
      IdsEmb);
  return p;
}

nvinfer1::IPluginV2* EmbLayerNormVarSeqlenPluginHFaceCreator::deserializePlugin(
    char const* name, void const* serialData, size_t serialLength) noexcept {
  return new EmbLayerNormVarSeqlenPluginHFace(name, serialData, serialLength);
}

nvinfer1::IPluginV2* EmbLayerNormVarSeqlenPluginMTronCreator::deserializePlugin(
    char const* name, void const* serialData, size_t serialLength) noexcept {
  return new EmbLayerNormVarSeqlenPluginMTron(name, serialData, serialLength);
}

void EmbLayerNormVarSeqlenPluginBaseCreator::setPluginNamespace(
    char const* libNamespace) noexcept {
  mNamespace = libNamespace;
}

char const* EmbLayerNormVarSeqlenPluginBaseCreator::getPluginNamespace()
    const noexcept {
  return mNamespace.c_str();
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
