#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include <vector>
#include "paddle/fluid/inference/tensorrt/plugin/special_slice_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

#if IS_TRT_VERSION_GE(6000)
SpecialSlicePluginDynamic::SpecialSlicePluginDynamic() {}

SpecialSlicePluginDynamic::SpecialSlicePluginDynamic(void const* serial_data,
                                                     size_t serial_length) {}

SpecialSlicePluginDynamic::~SpecialSlicePluginDynamic() {}

nvinfer1::IPluginV2DynamicExt* SpecialSlicePluginDynamic::clone() const {
  return new SpecialSlicePluginDynamic();
}

const char* SpecialSlicePluginDynamic::getPluginType() const {
  return "special_slice_plugin";
}

int SpecialSlicePluginDynamic::getNbOutputs() const { return 1; }

int SpecialSlicePluginDynamic::initialize() { return 0; }

size_t SpecialSlicePluginDynamic::getSerializationSize() const {
  size_t serialize_size = 0;
  return serialize_size;
}

void SpecialSlicePluginDynamic::serialize(void* buffer) const {}

nvinfer1::DimsExprs SpecialSlicePluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs* inputs, int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) {
  nvinfer1::DimsExprs output(inputs[0]);
  auto one = expr_builder.constant(1);
  output.d[0] = expr_builder.operation(nvinfer1::DimensionOperation::kSUB,
                                       *inputs[1].d[0], *one);

  return output;
}

void SpecialSlicePluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs) {}

size_t SpecialSlicePluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const {
  return 0;
}

void SpecialSlicePluginDynamic::destroy() { delete this; }

void SpecialSlicePluginDynamic::terminate() {}

bool SpecialSlicePluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* desc, int nb_inputs,
    int nb_outputs) {
  if (pos == 0)  // slice tensor
    return (desc[pos].type == nvinfer1::DataType::kHALF &&
            desc[pos].format ==
                nvinfer1::TensorFormat::kLINEAR);  // || desc[pos].type ==
  // nvinfer1::DataType::kFLOAT);

  if (pos == 1)  // cu_seqlen
    return (desc[pos].type == nvinfer1::DataType::kINT32 &&
            desc[pos].format == nvinfer1::TensorFormat::kLINEAR);

  return (desc[pos].type == nvinfer1::DataType::kHALF &&
          desc[pos].format ==
              nvinfer1::TensorFormat::kLINEAR);  // || desc[pos].type ==
  // nvinfer1::DataType::kFLOAT);
}

nvinfer1::DataType SpecialSlicePluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The index should be equal to 0"));
  return input_types[0];
}

template <typename T>
__global__ void SpecialSliceKernel(const T* slice_input,
                                   const int32_t* cu_seqlens, T* output) {
  const int hidden = blockDim.x;
  const int batch = blockIdx.x;

  output[batch * hidden + threadIdx.x] =
      slice_input[cu_seqlens[batch] * hidden + threadIdx.x];
}

int SpecialSlicePluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc, const void* const* inputs,
    void* const* outputs, void* workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;  // (sum(S), 768, 1, 1)
  auto out_dims = output_desc[0].dims;   // (batch, 768, 1, 1)

  assert(input_desc[0].type == nvinfer1::DataType::kHALF);

  const int32_t hidden = input_dims.d[1];
  const int num_blocks = out_dims.d[0];  // batch size
  const int num_threads = hidden;

  const half* slice_input = static_cast<const half*>(inputs[0]);
  const int32_t* cu_seqlens = static_cast<const int32_t*>(inputs[1]);
  half* output = static_cast<half*>(outputs[0]);

  SpecialSliceKernel<<<num_blocks, num_threads, 0, stream>>>(
      slice_input, cu_seqlens, output);

  return hipGetLastError() != hipSuccess;
}

SpecialSlicePluginDynamicCreator::SpecialSlicePluginDynamicCreator() {}

const char* SpecialSlicePluginDynamicCreator::getPluginName() const {
  return "special_slice_plugin";
}

const char* SpecialSlicePluginDynamicCreator::getPluginVersion() const {
  return "1";
}

const nvinfer1::PluginFieldCollection*
SpecialSlicePluginDynamicCreator::getFieldNames() {
  return &field_collection_;
}

nvinfer1::IPluginV2* SpecialSlicePluginDynamicCreator::createPlugin(
    const char* name, const nvinfer1::PluginFieldCollection* fc) {
  return new SpecialSlicePluginDynamic();
}

nvinfer1::IPluginV2* SpecialSlicePluginDynamicCreator::deserializePlugin(
    const char* name, const void* serial_data, size_t serial_length) {
  auto plugin = new SpecialSlicePluginDynamic(serial_data, serial_length);
  return plugin;
}

void SpecialSlicePluginDynamicCreator::setPluginNamespace(
    const char* lib_namespace) {
  plugin_namespace_ = lib_namespace;
}

const char* SpecialSlicePluginDynamicCreator::getPluginNamespace() const {
  return plugin_namespace_.c_str();
}

#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
