#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <cstring>
#include <vector>
#include "paddle/fluid/inference/tensorrt/plugin/special_slice_plugin.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

#if IS_TRT_VERSION_GE(6000)
SpecialSlicePluginDynamic::SpecialSlicePluginDynamic() {}

SpecialSlicePluginDynamic::SpecialSlicePluginDynamic(void const* serial_data,
                                                     size_t serial_length) {}

SpecialSlicePluginDynamic::~SpecialSlicePluginDynamic() {}

nvinfer1::IPluginV2DynamicExt* SpecialSlicePluginDynamic::clone() const
    TRT_NOEXCEPT {
  return new SpecialSlicePluginDynamic();
}

const char* SpecialSlicePluginDynamic::getPluginType() const TRT_NOEXCEPT {
  return "special_slice_plugin";
}

int SpecialSlicePluginDynamic::getNbOutputs() const TRT_NOEXCEPT { return 1; }

int SpecialSlicePluginDynamic::initialize() TRT_NOEXCEPT { return 0; }

size_t SpecialSlicePluginDynamic::getSerializationSize() const TRT_NOEXCEPT {
  size_t serialize_size = 0;
  return serialize_size;
}

void SpecialSlicePluginDynamic::serialize(void* buffer) const TRT_NOEXCEPT {}

nvinfer1::DimsExprs SpecialSlicePluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs* inputs, int nb_inputs,
    nvinfer1::IExprBuilder& expr_builder) TRT_NOEXCEPT {
  nvinfer1::DimsExprs output(inputs[0]);
  output.nbDims++;
  for (int i = output.nbDims - 1; i > 1; i--) {
    output.d[i] = inputs[0].d[i - 1];
  }
  auto one = expr_builder.constant(1);
  output.d[1] = one;
  output.d[0] = expr_builder.operation(nvinfer1::DimensionOperation::kSUB,
                                       *inputs[1].d[0], *one);
  // remove padding 1
  output.nbDims -= 2;

  return output;
}

void SpecialSlicePluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs) TRT_NOEXCEPT {}

size_t SpecialSlicePluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs,
    int nbOutputs) const TRT_NOEXCEPT {
  return 0;
}

void SpecialSlicePluginDynamic::destroy() TRT_NOEXCEPT { delete this; }

void SpecialSlicePluginDynamic::terminate() TRT_NOEXCEPT {}

bool SpecialSlicePluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc* desc, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  if (pos == 0)  // slice tensor
    return (desc[pos].type == nvinfer1::DataType::kHALF &&
            desc[pos].format ==
                nvinfer1::TensorFormat::kLINEAR);  // || desc[pos].type ==
  // nvinfer1::DataType::kFLOAT);

  if (pos == 1)  // cu_seqlen
    return (desc[pos].type == nvinfer1::DataType::kINT32 &&
            desc[pos].format == nvinfer1::TensorFormat::kLINEAR);

  return (desc[pos].type == nvinfer1::DataType::kHALF &&
          desc[pos].format ==
              nvinfer1::TensorFormat::kLINEAR);  // || desc[pos].type ==
  // nvinfer1::DataType::kFLOAT);
}

nvinfer1::DataType SpecialSlicePluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The index should be equal to 0"));
  return input_types[0];
}

template <typename T>
__global__ void SpecialSliceKernel(const T* slice_input,
                                   const int32_t* cu_seqlens, T* output) {
  const int hidden = blockDim.x * gridDim.x;
  const int hidden_id = blockIdx.x * blockDim.x + threadIdx.x;
  const int batch_id = blockIdx.y;

  output[batch_id * hidden + hidden_id] =
      slice_input[cu_seqlens[batch_id] * hidden + hidden_id];
}

int SpecialSlicePluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc* input_desc,
    const nvinfer1::PluginTensorDesc* output_desc, const void* const* inputs,
    void* const* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;  // (sum(S), hidden, 1, 1)
  auto out_dims = output_desc[0].dims;   // (batch, hidden, 1, 1)

  PADDLE_ENFORCE_EQ(
      input_desc[0].type, nvinfer1::DataType::kHALF,
      platform::errors::InvalidArgument("Type of input should be half."));

  const int32_t hidden = input_dims.d[1];
  PADDLE_ENFORCE_EQ(hidden % 128, 0, platform::errors::InvalidArgument(
                                         "hidden should be multiple of 128."));

  constexpr int num_threads = 128;
  const half* slice_input = static_cast<const half*>(inputs[0]);
  const int32_t* cu_seqlens = static_cast<const int32_t*>(inputs[1]);
  half* output = static_cast<half*>(outputs[0]);

  const int32_t num_blocks_x = hidden / num_threads;
  const int32_t num_blocks_y = out_dims.d[0];         // batchs
  const dim3 num_blocks(num_blocks_x, num_blocks_y);  // blocks

  SpecialSliceKernel<<<num_blocks, num_threads, 0, stream>>>(
      slice_input, cu_seqlens, output);
  return hipGetLastError() != hipSuccess;
}

SpecialSlicePluginDynamicCreator::SpecialSlicePluginDynamicCreator() {}

const char* SpecialSlicePluginDynamicCreator::getPluginName() const
    TRT_NOEXCEPT {
  return "special_slice_plugin";
}

const char* SpecialSlicePluginDynamicCreator::getPluginVersion() const
    TRT_NOEXCEPT {
  return "1";
}

const nvinfer1::PluginFieldCollection*
SpecialSlicePluginDynamicCreator::getFieldNames() TRT_NOEXCEPT {
  return &field_collection_;
}

nvinfer1::IPluginV2* SpecialSlicePluginDynamicCreator::createPlugin(
    const char* name, const nvinfer1::PluginFieldCollection* fc) TRT_NOEXCEPT {
  return new SpecialSlicePluginDynamic();
}

nvinfer1::IPluginV2* SpecialSlicePluginDynamicCreator::deserializePlugin(
    const char* name, const void* serial_data,
    size_t serial_length) TRT_NOEXCEPT {
  auto plugin = new SpecialSlicePluginDynamic(serial_data, serial_length);
  return plugin;
}

void SpecialSlicePluginDynamicCreator::setPluginNamespace(
    const char* lib_namespace) TRT_NOEXCEPT {
  plugin_namespace_ = lib_namespace;
}

const char* SpecialSlicePluginDynamicCreator::getPluginNamespace() const
    TRT_NOEXCEPT {
  return plugin_namespace_.c_str();
}

#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
