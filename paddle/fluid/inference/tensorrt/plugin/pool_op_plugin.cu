#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/pool_op_plugin.h"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_factory.h"
#include "paddle/fluid/operators/math/pooling.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

PoolPlugin *CreatePoolPluginDeserialize(const void *buffer, size_t length) {
  return new PoolPlugin(buffer, length);
}
REGISTER_TRT_PLUGIN("pool_plugin", CreatePoolPluginDeserialize);

nvinfer1::Dims PoolPlugin::getOutputDimensions(int index,
                                               const nvinfer1::Dims *inputDims,
                                               int nbInputs) {
  assert(nbInputs == 1);
  assert(index == 0);
  assert(inputDims[0].nbDims == 3);
  nvinfer1::Dims const &input_dims = inputDims[0];

  nvinfer1::Dims output_dims = input_dims;

  output_dims.d[1] = output_shape_[1];
  output_dims.d[2] = output_shape_[2];
  return output_dims;
}

int PoolPlugin::enqueue(int batchSize, const void *const *inputs,
                        void **outputs, void *workspace, hipStream_t stream) {
  auto const &input_dims = this->getInputDims(0);
  int input_size = 0;
  float const *idata = reinterpret_cast<float const *>(inputs[0]);
  float **odatas = reinterpret_cast<float **>(outputs);

  std::vector<int> input_shape = input_shape_;
  std::vector<int> output_shape = output_shape_;
  input_shape.insert(input_shape.begin(), batchSize);
  output_shape.insert(output_shape.begin(), batchSize);

  if (pool_type_ == PoolType::max) {
    paddle::operators::math::MaxPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::MaxPool<float>, float>
        pool2d_forward;
    pool2d_forward(idata, input_shape, output_shape, ksize_, strides_,
                   paddings_, pool_process, true, adaptive_, odatas[0], stream);
  } else if (pool_type_ == PoolType::avg) {
    paddle::operators::math::AvgPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::AvgPool<float>, float>
        pool2d_forward;
    pool2d_forward(idata, input_shape, output_shape, ksize_, strides_,
                   paddings_, pool_process, true, adaptive_, odatas[0], stream);
  }

  return hipGetLastError() != hipSuccess;
}

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

size_t PoolPluginDynamic::getSerializationSize() const { return 0; }

void PoolPluginDynamic::serialize(void *buffer) const {}

nvinfer1::DimsExprs PoolPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) {
  PADDLE_ENFORCE_EQ(nb_inputs, 1,
                    platform::errors::InvalidArgument(
                        "The Split plugin should be only one input."));

  PADDLE_ENFORCE_EQ(
      inputs[0].d[1]->isConstant(), true,
      platform::errors::InvalidArgument("The channel dimension should be "
                                        "static, but we found it's dynamic."));
  nvinfer1::DimsExprs output(inputs[0]);
  if (is_global_) {
    output.d[2] = expr_builder.constant(1);
    output.d[3] = expr_builder.constant(1);
    return output;
  }
  if (adaptive_) {
    output.d[2] = expr_builder.constant(ksize_[0]);
    output.d[3] = expr_builder.constant(ksize_[1]);
    return output;
  }

  auto stri_0 = expr_builder.constant(strides_[0]);
  auto stri_1 = expr_builder.constant(strides_[1]);

  auto tmp1_0 =
      expr_builder.constant((-ksize_[0] + 2 * paddings_[0]) / strides_[0] + 1);
  auto tmp1_1 =
      expr_builder.constant((-ksize_[1] + 2 * paddings_[1]) / strides_[1] + 1);

  auto tmp2_0 = expr_builder.constant(
      (-ksize_[0] + 2 * paddings_[0] + strides_[0] - 1) / strides_[0] + 1);
  auto tmp2_1 = expr_builder.constant(
      (-ksize_[1] + 2 * paddings_[1] + strides_[1] - 1) / strides_[1] + 1);

  auto *a_d = expr_builder.operation(nvinfer1::DimensionOperation::kCEIL_DIV,
                                     *inputs[0].d[2], *stri_0);
  auto *b_d = expr_builder.operation(nvinfer1::DimensionOperation::kCEIL_DIV,
                                     *inputs[0].d[3], *stri_1);

  if (!ceil_mode_) {
    output.d[2] = expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                         *a_d, *tmp1_0);
    output.d[3] = expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                         *b_d, *tmp1_1);
  } else {
    output.d[2] = expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                         *a_d, *tmp2_0);
    output.d[3] = expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                         *b_d, *tmp2_1);
  }

  return output;
}

bool PoolPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  return ((in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
          in_out[pos].format == nvinfer1::PluginFormat::kNCHW);
}

nvinfer1::DataType PoolPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types, int nb_inputs) const {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The Pool Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT), true,
                    platform::errors::InvalidArgument(
                        "The input type should be half or float"));
  return input_types[0];
}

int PoolPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                               const nvinfer1::PluginTensorDesc *output_desc,
                               const void *const *inputs, void *const *outputs,
                               void *workspace, hipStream_t stream) {
  auto input_dims = input_desc[0].dims;
  int n = input_dims.d[0];
  int c = input_dims.d[1];
  int h = input_dims.d[2];
  int w = input_dims.d[3];

  const float *input = static_cast<const float *>(inputs[0]);
  float *output = static_cast<float *>(outputs[0]);

  std::vector<int> input_shape, output_shape;
  for (int i = 0; i < input_dims.nbDims; i++)
    input_shape.push_back(input_dims.d[i]);
  output_shape = input_shape;

  std::vector<int> ksize = ksize_;
  std::vector<int> paddings = paddings_;
  if (is_global_) {
    ksize[0] = h;
    ksize[1] = w;
    paddings[0] = 0;
    paddings[1] = 0;
    output_shape[2] = 1;
    output_shape[3] = 1;
  } else {
    auto data_dim = CalcOutputSize({h, w}, ceil_mode_, adaptive_, ksize_,
                                   strides_, paddings_);
    output_shape[2] = data_dim[0];
    output_shape[3] = data_dim[1];
  }

  if (pool_type_ == "max") {
    paddle::operators::math::MaxPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::MaxPool<float>, float>
        pool2d_forward;
    pool2d_forward(input, input_shape, output_shape, ksize, strides_, paddings,
                   pool_process, true, adaptive_, output, stream);
  } else if (pool_type_ == "avg") {
    paddle::operators::math::AvgPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::AvgPool<float>, float>
        pool2d_forward;
    pool2d_forward(input, input_shape, output_shape, ksize, strides_, paddings,
                   pool_process, true, adaptive_, output, stream);
  }

  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
