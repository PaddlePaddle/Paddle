#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/pool_op_plugin.h"
#include "paddle/fluid/operators/math/pooling.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

nvinfer1::Dims PoolPlugin::getOutputDimensions(int index,
                                               const nvinfer1::Dims *inputDims,
                                               int nbInputs) TRT_NOEXCEPT {
  assert(nbInputs == 1);
  assert(index == 0);
  assert(inputDims[0].nbDims == 3);
  nvinfer1::Dims const &input_dims = inputDims[0];

  nvinfer1::Dims output_dims = input_dims;

  output_dims.d[1] = output_shape_[1];
  output_dims.d[2] = output_shape_[2];
  return output_dims;
}

size_t PoolPlugin::getSerializationSize() const TRT_NOEXCEPT {
  return getBaseSerializationSize() + SerializedSize(ceil_mode_) +
         SerializedSize(pool_type_) + SerializedSize(adaptive_) +
         SerializedSize(exclusive_) + SerializedSize(ksize_) +
         SerializedSize(strides_) + SerializedSize(paddings_) +
         SerializedSize(real_paddings_) + SerializedSize(input_shape_) +
         SerializedSize(output_shape_);
}

// TRT will call this func when we need to serialize the configuration of
// tensorrt.
void PoolPlugin::serialize(void *buffer) const TRT_NOEXCEPT {
  serializeBase(buffer);
  SerializeValue(&buffer, ceil_mode_);
  SerializeValue(&buffer, pool_type_);
  SerializeValue(&buffer, adaptive_);
  SerializeValue(&buffer, exclusive_);
  SerializeValue(&buffer, ksize_);
  SerializeValue(&buffer, strides_);
  SerializeValue(&buffer, paddings_);
  SerializeValue(&buffer, real_paddings_);
  SerializeValue(&buffer, input_shape_);
  SerializeValue(&buffer, output_shape_);
}

PoolPlugin *PoolPlugin::clone() const TRT_NOEXCEPT {
  return new PoolPlugin(ceil_mode_, pool_type_, adaptive_, exclusive_, ksize_,
                        strides_, paddings_, input_shape_, real_paddings_);
}

int PoolPlugin::enqueue(int batchSize, const void *const *inputs,
#if IS_TRT_VERSION_LT(8000)
                        void **outputs, void *workspace,
                        hipStream_t stream) TRT_NOEXCEPT {
#else
                        void *const *outputs, void *workspace,
                        hipStream_t stream) TRT_NOEXCEPT {
#endif
  auto const &input_dims = this->getInputDims(0);
  int input_size = 0;
  float const *idata = reinterpret_cast<float const *>(inputs[0]);
  float *const *odatas = reinterpret_cast<float *const *>(outputs);

  std::vector<int> input_shape = input_shape_;
  std::vector<int> output_shape = output_shape_;
  input_shape.insert(input_shape.begin(), batchSize);
  output_shape.insert(output_shape.begin(), batchSize);

  if (pool_type_ == PoolType::max) {
    paddle::operators::math::MaxPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::MaxPool<float>, float>
        pool2d_forward;
    pool2d_forward(idata, input_shape, output_shape, ksize_, strides_,
                   paddings_, true, false, odatas[0], stream, pool_process);
  } else if (pool_type_ == PoolType::avg) {
    paddle::operators::math::AvgPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::AvgPool<float>, float>
        pool2d_forward;
    pool2d_forward(idata, input_shape, output_shape, ksize_, strides_,
                   paddings_, exclusive_, adaptive_, odatas[0], stream,
                   pool_process);
  }

  return hipGetLastError() != hipSuccess;
}

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

PoolPluginDynamic::PoolPluginDynamic(void const *serialData,
                                     size_t serialLength) {
  DeserializeValue(&serialData, &serialLength, &ceil_mode_);
  const char *pool_type;
  DeserializeValue(&serialData, &serialLength, &pool_type);
  pool_type_ = std::string(pool_type);
  DeserializeValue(&serialData, &serialLength, &adaptive_);
  DeserializeValue(&serialData, &serialLength, &exclusive_);
  DeserializeValue(&serialData, &serialLength, &ksize_);
  DeserializeValue(&serialData, &serialLength, &strides_);
  DeserializeValue(&serialData, &serialLength, &paddings_);
  DeserializeValue(&serialData, &serialLength, &is_global_);
}

size_t PoolPluginDynamic::getSerializationSize() const TRT_NOEXCEPT {
  return SerializedSize(ceil_mode_) + SerializedSize(pool_type_.c_str()) +
         SerializedSize(adaptive_) + SerializedSize(exclusive_) +
         SerializedSize(ksize_) + SerializedSize(strides_) +
         SerializedSize(paddings_) + SerializedSize(is_global_);
}

void PoolPluginDynamic::serialize(void *buffer) const TRT_NOEXCEPT {
  SerializeValue(&buffer, ceil_mode_);
  SerializeValue(&buffer, pool_type_.c_str());
  SerializeValue(&buffer, adaptive_);
  SerializeValue(&buffer, exclusive_);
  SerializeValue(&buffer, ksize_);
  SerializeValue(&buffer, strides_);
  SerializeValue(&buffer, paddings_);
  SerializeValue(&buffer, is_global_);
}

nvinfer1::IPluginV2DynamicExt *PoolPluginDynamic::clone() const TRT_NOEXCEPT {
  return new PoolPluginDynamic(ceil_mode_, pool_type_, adaptive_, exclusive_,
                               ksize_, strides_, paddings_, is_global_);
}

nvinfer1::DimsExprs PoolPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(nb_inputs, 1,
                    platform::errors::InvalidArgument(
                        "The Split plugin should be only one input."));

  PADDLE_ENFORCE_EQ(
      inputs[0].d[1]->isConstant(), true,
      platform::errors::InvalidArgument("The channel dimension should be "
                                        "static, but we found it's dynamic."));
  nvinfer1::DimsExprs output(inputs[0]);
  if (is_global_ && !adaptive_) {
    output.d[2] = expr_builder.constant(1);
    output.d[3] = expr_builder.constant(1);
    return output;
  }
  if (is_global_ && adaptive_) {
    return inputs[0];
  }
  if (adaptive_) {
    output.d[2] = expr_builder.constant(ksize_[0]);
    output.d[3] = expr_builder.constant(ksize_[1]);
    return output;
  }

  auto stri_0 = expr_builder.constant(strides_[0]);
  auto stri_1 = expr_builder.constant(strides_[1]);
  auto one_value = expr_builder.constant(1);

  auto v0_tmp = expr_builder.constant(-ksize_[0] + 2 * paddings_[0]);
  auto v1_tmp = expr_builder.constant(-ksize_[1] + 2 * paddings_[1]);

  auto ceil_tmp =
      expr_builder.constant(-ksize_[0] + 2 * paddings_[0] + strides_[0] - 1);
  auto ceil1_tmp =
      expr_builder.constant(-ksize_[1] + 2 * paddings_[1] + strides_[1] - 1);

  if (!ceil_mode_) {
    output.d[2] = expr_builder.operation(
        nvinfer1::DimensionOperation::kSUM,
        *expr_builder.operation(
            nvinfer1::DimensionOperation::kFLOOR_DIV,
            *expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                    *inputs[0].d[2], *v0_tmp),
            *stri_0),
        *one_value);
    output.d[3] = expr_builder.operation(
        nvinfer1::DimensionOperation::kSUM,
        *expr_builder.operation(
            nvinfer1::DimensionOperation::kFLOOR_DIV,
            *expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                    *inputs[0].d[3], *v1_tmp),
            *stri_1),
        *one_value);

  } else {
    output.d[2] = expr_builder.operation(
        nvinfer1::DimensionOperation::kSUM,
        *expr_builder.operation(
            nvinfer1::DimensionOperation::kFLOOR_DIV,
            *expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                    *inputs[0].d[2], *ceil_tmp),
            *stri_0),
        *one_value);
    output.d[3] = expr_builder.operation(
        nvinfer1::DimensionOperation::kSUM,
        *expr_builder.operation(
            nvinfer1::DimensionOperation::kFLOOR_DIV,
            *expr_builder.operation(nvinfer1::DimensionOperation::kSUM,
                                    *inputs[0].d[3], *ceil1_tmp),
            *stri_1),
        *one_value);
  }

  return output;
}

bool PoolPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));
  (in_out && pos < (nb_inputs + nb_outputs));

  return ((in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
          in_out[pos].format == nvinfer1::PluginFormat::kLINEAR);
}

nvinfer1::DataType PoolPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(index, 0, platform::errors::InvalidArgument(
                                  "The Pool Plugin only has one input, so the "
                                  "index value should be 0, but get %d.",
                                  index));
  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kFLOAT), true,
                    platform::errors::InvalidArgument(
                        "The input type should be half or float"));
  return input_types[0];
}

int PoolPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *input_desc,
                               const nvinfer1::PluginTensorDesc *output_desc,
                               const void *const *inputs, void *const *outputs,
                               void *workspace,
                               hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  int n = input_dims.d[0];
  int c = input_dims.d[1];
  int h = input_dims.d[2];
  int w = input_dims.d[3];

  const float *input = static_cast<const float *>(inputs[0]);
  float *output = static_cast<float *>(outputs[0]);

  std::vector<int> input_shape, output_shape;
  for (int i = 0; i < input_dims.nbDims; i++)
    input_shape.push_back(input_dims.d[i]);
  output_shape = input_shape;

  std::vector<int> ksize = ksize_;
  std::vector<int> paddings = paddings_;
  if (is_global_) {
    ksize[0] = h;
    ksize[1] = w;
    paddings[0] = 0;
    paddings[1] = 0;
    output_shape[2] = 1;
    output_shape[3] = 1;
  } else {
    auto data_dim = CalcOutputSize({h, w}, ceil_mode_, adaptive_, ksize_,
                                   strides_, paddings_);
    output_shape[2] = data_dim[0];
    output_shape[3] = data_dim[1];
  }
  if (adaptive_) {
    output_shape[2] = h;
    output_shape[3] = w;
  }

  if (pool_type_ == "max") {
    paddle::operators::math::MaxPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::MaxPool<float>, float>
        pool2d_forward;
    pool2d_forward(input, input_shape, output_shape, ksize, strides_, paddings,
                   true, false, output, stream, pool_process);
  } else if (pool_type_ == "avg") {
    paddle::operators::math::AvgPool<float> pool_process;
    paddle::operators::math::Pool2dDirectCUDAFunctor<
        paddle::operators::math::AvgPool<float>, float>
        pool2d_forward;
    pool2d_forward(input, input_shape, output_shape, ksize, strides_, paddings,
                   exclusive_, adaptive_, output, stream, pool_process);
  }

  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
