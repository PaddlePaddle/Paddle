#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
// Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>

#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/preln_residual_bias_plugin.h"
#include "paddle/fluid/operators/fused/fused_dropout_common.h"
#include "paddle/fluid/operators/fused/fused_layernorm_residual_dropout_bias.h"
#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
#define FINAL_MASK 0xffffffff
template <typename T, int NUM>
__inline__ __device__ T warpReduceSumV2(T *val) {
#pragma unroll
  for (int i = 0; i < NUM; i++) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
      val[i] += __shfl_xor_sync(FINAL_MASK, val[i], mask, 32);
  }
  return (T)(0.0f);
}

template <typename T, int NUM>
__inline__ __device__ T blockReduceSumV2(T *val) {
  static __shared__ T shared[NUM][33];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  warpReduceSumV2<T, NUM>(val);

  if (lane == 0) {
#pragma unroll
    for (int i = 0; i < NUM; i++) {
      shared[i][wid] = val[i];
    }
  }
  __syncthreads();

  bool is_mask = threadIdx.x < (blockDim.x / 32.f);
#pragma unroll
  for (int i = 0; i < NUM; i++) {
    val[i] = is_mask ? shared[i][lane] : (T)(0.0f);
  }
  warpReduceSumV2<T, NUM>(val);
  return (T)0.0f;
}

__global__ void generalAddBiasResidualLayerNormOpt2(
    half2 *normed_output,
    half2 *output,
    const half2 *__restrict bias,
    const half2 *__restrict src,
    const half2 *__restrict residual,
    const half2 *__restrict gamma,
    const half2 *__restrict beta,
    int m,
    int n,
    float epsilon) {
  __shared__ float s_mean;
  __shared__ float s_variance;
  float x_sum = 0.0f;
  float x2_sum = 0.0f;
  const int b_offset = blockIdx.x * n;

#pragma unroll 2
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int index = b_offset + i;
    float val_1 = 0.0f;
    float val_2 = 0.0f;
    half2 tmp;

    if (bias) {
      tmp = __ldg(&bias[i]);
      val_1 += static_cast<float>(tmp.x);
      val_2 += static_cast<float>(tmp.y);
    }
    {
      tmp = __ldg(&residual[index]);
      val_1 += static_cast<float>(tmp.x);
      val_2 += static_cast<float>(tmp.y);
    }
    {
      tmp = __ldg(&src[index]);
      val_1 += static_cast<float>(tmp.x);
      val_2 += static_cast<float>(tmp.y);
    }
    tmp.x = __float2half_rn(val_1);
    tmp.y = __float2half_rn(val_2);
    output[index] = tmp;
    x_sum += val_1 + val_2;
    x2_sum += val_1 * val_1 + val_2 * val_2;
  }
  float sums[2];
  sums[0] = x_sum;
  sums[1] = x2_sum;
  blockReduceSumV2<float, 2>(sums);

  if (threadIdx.x == 0) {
    s_mean = sums[0] / n / 2;
    s_variance = rsqrtf(sums[1] / n / 2 - s_mean * s_mean + epsilon);
  }
  __syncthreads();

  half2 mean_2 = __float2half2_rn(s_mean);
  half2 var_2 = __float2half2_rn(s_variance);

#pragma unroll 2
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int index = b_offset + i;
    half2 val = __hmul2(__hmul2(__hsub2(output[index], mean_2), var_2),
                        __ldg(&gamma[i]));
    if (beta) {
      val = __hadd2(val, __ldg(&beta[i]));
    }
    normed_output[index] = val;
  }
}
#endif

using half = phi::dtype::float16;

#if IS_TRT_VERSION_GE(6000)
int PrelnResidualBiasPluginDynamic::initialize() TRT_NOEXCEPT {
  hipMalloc(&bias_gpu_, sizeof(float) * bias_size_);
  hipMemcpy(bias_gpu_,
             bias_.data(),
             bias_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&scale_gpu_, sizeof(float) * scale_size_);
  hipMemcpy(scale_gpu_,
             scale_.data(),
             scale_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  if (ele_bias_size_ > 0) {
    if (with_fp16_) {
      hipMalloc(&ele_bias_gpu_, sizeof(half) * ele_bias_size_);
      hipMemcpy(ele_bias_gpu_,
                 fp16_ele_bias_.data(),
                 ele_bias_size_ * sizeof(half),
                 hipMemcpyHostToDevice);
    } else {
      hipMalloc(&ele_bias_gpu_, sizeof(float) * ele_bias_size_);
      hipMemcpy(ele_bias_gpu_,
                 fp32_ele_bias_.data(),
                 ele_bias_size_ * sizeof(float),
                 hipMemcpyHostToDevice);
    }
  } else {
    ele_bias_gpu_ = nullptr;
  }

  return 0;
}

void PrelnResidualBiasPluginDynamic::terminate() TRT_NOEXCEPT {
  if (bias_gpu_) {
    hipFree(bias_gpu_);
    bias_gpu_ = nullptr;
  }
  if (scale_gpu_) {
    hipFree(scale_gpu_);
    scale_gpu_ = nullptr;
  }
  if (ele_bias_gpu_) {
    hipFree(ele_bias_gpu_);
    ele_bias_gpu_ = nullptr;
  }
}

nvinfer1::IPluginV2DynamicExt *PrelnResidualBiasPluginDynamic::clone() const
    TRT_NOEXCEPT {
  PrelnResidualBiasPluginDynamic *ptr = nullptr;
  if (with_fp16_) {
    ptr = new PrelnResidualBiasPluginDynamic(bias_.data(),
                                             scale_.data(),
                                             fp16_ele_bias_.data(),
                                             bias_size_,
                                             scale_size_,
                                             ele_bias_size_,
                                             eps_,
                                             with_fp16_);
  } else {
    ptr = new PrelnResidualBiasPluginDynamic(bias_.data(),
                                             scale_.data(),
                                             fp32_ele_bias_.data(),
                                             bias_size_,
                                             scale_size_,
                                             ele_bias_size_,
                                             eps_,
                                             with_fp16_);
  }

  ptr->bias_gpu_ = bias_gpu_;
  ptr->scale_gpu_ = scale_gpu_;
  ptr->ele_bias_gpu_ = ele_bias_gpu_;
  return ptr;
}

const char *PrelnResidualBiasPluginDynamic::getPluginType() const TRT_NOEXCEPT {
  return "preln_residual_bias_plugin_dynamic";
}

int PrelnResidualBiasPluginDynamic::getNbOutputs() const TRT_NOEXCEPT {
  return 2;
}

size_t PrelnResidualBiasPluginDynamic::getSerializationSize() const
    TRT_NOEXCEPT {
  size_t ser_size = SerializedSize(bias_) + SerializedSize(scale_) +
                    SerializedSize(fp32_ele_bias_) +
                    SerializedSize(fp16_ele_bias_) +
                    SerializedSize(bias_size_) + SerializedSize(scale_size_) +
                    SerializedSize(ele_bias_size_) + SerializedSize(eps_) +
                    SerializedSize(with_fp16_);
  return ser_size;
}
void PrelnResidualBiasPluginDynamic::serialize(void *buffer) const
    TRT_NOEXCEPT {
  SerializeValue(&buffer, bias_);
  SerializeValue(&buffer, scale_);
  SerializeValue(&buffer, fp32_ele_bias_);
  SerializeValue(&buffer, fp16_ele_bias_);
  SerializeValue(&buffer, bias_size_);
  SerializeValue(&buffer, scale_size_);
  SerializeValue(&buffer, ele_bias_size_);
  SerializeValue(&buffer, eps_);
  SerializeValue(&buffer, with_fp16_);
}

nvinfer1::DimsExprs PrelnResidualBiasPluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  if (output_index < 2) {
    return inputs[0];
  } else {  // moving mean and var
    nvinfer1::DimsExprs ret;
    ret.nbDims = 1;
    ret.d[0] = inputs[0].d[2];
    return ret;
  }
}

bool PrelnResidualBiasPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
      PADDLE_THROW(
          platform::errors::Fatal("TRT plugin supported FP16 is not available "
                                  "while with_fp16 is set true."));
#endif
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

void PrelnResidualBiasPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc *in,
    int nb_inputs,
    const nvinfer1::DynamicPluginTensorDesc *out,
    int nb_outputs) TRT_NOEXCEPT {}

size_t PrelnResidualBiasPluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc *inputs,
    int nb_inputs,
    const nvinfer1::PluginTensorDesc *outputs,
    int nb_outputs) const TRT_NOEXCEPT {
  return 0;
}

nvinfer1::DataType PrelnResidualBiasPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

void PrelnResidualBiasPluginDynamic::destroy() TRT_NOEXCEPT { delete this; }

int PrelnResidualBiasPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  int hidden = input_dims.d[2];
  const size_t rows = static_cast<size_t>(
      input_dims.d[0] * input_dims.d[1]);  // batch * seq_length
  const size_t cols = static_cast<size_t>(input_dims.d[2]);

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp32";
    const float *input1 = static_cast<const float *>(inputs[0]);
    const float *input2 = static_cast<const float *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train = false;
    const bool is_test = true;
    const uint64_t increment = 0;
    const float epsilon = eps_;
    const float *src = input2;
    const float *residual = input1;
    const float *bias = static_cast<float *>(ele_bias_gpu_);
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data = nullptr;
    float *dst = static_cast<float *>(outputs[1]);
    float *layernorm_dst = static_cast<float *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;
    const int VecSize = 8;
    paddle::operators::FusedLayernormResidualDropoutBiasFunctor<float,
                                                                uint8_t,
                                                                VecSize,
                                                                float,
                                                                false>()(
        rows,
        cols,
        seed,
        dropout_prob,
        is_upscale_in_train,
        is_test,
        increment,
        epsilon,
        src,
        residual,
        bias,
        scale,
        layernorm_bias,
        mask_data,
        dst,
        layernorm_dst,
        mean,
        var,
        stream);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp16";
    const half *input1 = static_cast<const half *>(inputs[0]);
    const half *input2 = static_cast<const half *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train = false;
    const bool is_test = true;
    const uint64_t increment = 0;
    const float epsilon = eps_;
    const half *src = input2;
    const half *residual = input1;
    const half *bias = static_cast<half *>(ele_bias_gpu_);
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data = nullptr;
    half *dst = static_cast<half *>(outputs[1]);
    half *layernorm_dst = static_cast<half *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;
    const int VecSize = 8;
    // if odd
    if (hidden & 1 == 0) {
      int half_n = hidden / 2;
      int half_n_32 = (half_n + 31) / 32 * 32;
      int block(std::min(half_n_32, 512));
      generalAddBiasResidualLayerNormOpt2<<<rows, block, 0, stream>>>(
          reinterpret_cast<half2 *>(layernorm_dst),
          reinterpret_cast<half2 *>(dst),
          (const half2 *)bias,
          (const half2 *)input2,
          (const half2 *)input1,
          (const half2 *)scale,
          (const half2 *)layernorm_bias,
          rows,
          half_n,
          epsilon);
    } else {
      paddle::operators::FusedLayernormResidualDropoutBiasFunctor<half,
                                                                  uint8_t,
                                                                  VecSize,
                                                                  float,
                                                                  false>()(
          rows,
          cols,
          seed,
          dropout_prob,
          is_upscale_in_train,
          is_test,
          increment,
          epsilon,
          src,
          residual,
          bias,
          scale,
          layernorm_bias,
          mask_data,
          dst,
          layernorm_dst,
          mean,
          var,
          stream);
    }
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) tensorRT plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The PrelnResidualBias TRT Plugin's input type "
                                "should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

const char *PrelnResidualBiasPluginDynamicCreator::getPluginName() const
    TRT_NOEXCEPT {
  return "preln_residual_bias_plugin_dynamic";
}

const char *PrelnResidualBiasPluginDynamicCreator::getPluginVersion() const
    TRT_NOEXCEPT {
  return "1";
}

nvinfer1::IPluginV2 *PrelnResidualBiasPluginDynamicCreator::deserializePlugin(
    const char *name,
    const void *serial_data,
    size_t serial_length) TRT_NOEXCEPT {
  return new PrelnResidualBiasPluginDynamic(serial_data, serial_length);
}

#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
