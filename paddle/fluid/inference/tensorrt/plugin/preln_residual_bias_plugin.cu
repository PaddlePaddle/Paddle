// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/preln_residual_bias_plugin.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"
#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/fluid/operators/fused/fused_dropout_common.h"
#include "paddle/fluid/operators/fused/fused_layernorm_residual_dropout_bias.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
using half = phi::dtype::float16;
// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

template<typename T>
int PrelnResidualBiasPluginDynamic<T>::initialize() TRT_NOEXCEPT {
  hipMalloc(&bias_gpu_, sizeof(float) * bias_size_);
  hipMemcpy(bias_gpu_, bias_.data(), bias_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&scale_gpu_, sizeof(float) * scale_size_);
  hipMemcpy(scale_gpu_, scale_.data(), scale_size_ * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&ele_bias_gpu_, sizeof(T) * ele_bias_size_);
  hipMemcpy(ele_bias_gpu_, ele_bias_.data(), ele_bias_size_ * sizeof(T),
             hipMemcpyHostToDevice);


  return 0;
}

template<typename T>
void PrelnResidualBiasPluginDynamic<T>::terminate() TRT_NOEXCEPT {
  if (bias_gpu_) {
    hipFree(bias_gpu_);
    bias_gpu_ = nullptr;
  }
  if (scale_gpu_) {
    hipFree(scale_gpu_);
    scale_gpu_ = nullptr;
  }
  if (ele_bias_gpu_) {
    hipFree(ele_bias_gpu_);
    ele_bias_gpu_ = nullptr;
  }
}

template<typename T>
nvinfer1::DimsExprs PrelnResidualBiasPluginDynamic<T>::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  return inputs[0];
}

template<typename T>
bool PrelnResidualBiasPluginDynamic<T>::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
    } else {
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

template<typename T>
nvinfer1::DataType PrelnResidualBiasPluginDynamic<T>::getOutputDataType(
    int index, const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
//  PADDLE_ENFORCE_EQ(index, 0,
//                    platform::errors::InvalidArgument(
//                        "The PrelnResidualBias Plugin only has one input, so the "
//                        "index value should be 0, but get %d.",
//                        index));
//  PADDLE_ENFORCE_EQ((input_types[0] == nvinfer1::DataType::kHALF),
//                    true, platform::errors::InvalidArgument(
//                              "The input type should be half or float"));
  return input_types[0];
}

template<typename T>
int PrelnResidualBiasPluginDynamic<T>::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  size_t num = ProductDim(input_dims);
  int hidden = input_dims.d[2];
  const size_t  rows = static_cast<size_t>(input_dims.d[0] * input_dims.d[1]); // batch * seq_length
  const size_t cols = static_cast<size_t>(input_dims.d[2]);

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp32";

    PADDLE_THROW(platform::errors::Fatal(
        "unsupported float format!!!"));

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp16";
    const half *input1 = static_cast<const half *>(inputs[0]);
    const half *input2 = static_cast<const half *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train=false;
    const bool is_test = true;
    const uint64_t increment=0;
    const float epsilon = eps_;
    const half *src = input2;
    const half *residual = input1;
    const half *bias = ele_bias_gpu_;
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data=nullptr;
    half *dst = static_cast<half *>(outputs[1]);;
    half *layernorm_dst = static_cast<half *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;

//    const int VecSize = paddle::operators::MAX_CACHE_BYTES / sizeof(T);
    const int VecSize = 8;
    paddle::operators::FusedLayernormResidualDropoutBiasFunctor<half, uint8_t, VecSize, float, false>()(
          rows, cols, seed, dropout_prob, is_upscale_in_train, is_test,
          increment, epsilon, src, residual, bias, scale, layernorm_bias,
          mask_data, dst, layernorm_dst, mean, var, stream);
//    hipDeviceSynchronize();
    VLOG(6) << "finish FusedLayernormResidualDropoutBiasFunctor";
//    operators::math::PrelnResidualBiasFunctor<half> skip_layer_norm_func;
//    skip_layer_norm_func(num, hidden, input1, input2, scale_gpu_, bias_gpu_,
//                         output, static_cast<half>(eps_), stream);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) tensorRT plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The PrelnResidualBias TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

template class PrelnResidualBiasPluginDynamic<half>; 
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
