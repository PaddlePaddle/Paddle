// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>

#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/preln_residual_bias_plugin.h"
#include "paddle/fluid/operators/fused/fused_dropout_common.h"
#include "paddle/fluid/operators/fused/fused_layernorm_residual_dropout_bias.h"
#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
using half = phi::dtype::float16;

#if IS_TRT_VERSION_GE(6000)
int PrelnResidualBiasPluginDynamic::initialize() TRT_NOEXCEPT {
  hipMalloc(&bias_gpu_, sizeof(float) * bias_size_);
  hipMemcpy(bias_gpu_, bias_.data(), bias_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&scale_gpu_, sizeof(float) * scale_size_);
  hipMemcpy(scale_gpu_, scale_.data(), scale_size_ * sizeof(float),
             hipMemcpyHostToDevice);

  if (with_fp16_) {
    hipMalloc(&ele_bias_gpu_, sizeof(half) * ele_bias_size_);
    hipMemcpy(ele_bias_gpu_, fp16_ele_bias_.data(),
               ele_bias_size_ * sizeof(half), hipMemcpyHostToDevice);
  } else {
    hipMalloc(&ele_bias_gpu_, sizeof(float) * ele_bias_size_);
    hipMemcpy(ele_bias_gpu_, fp32_ele_bias_.data(),
               ele_bias_size_ * sizeof(float), hipMemcpyHostToDevice);
  }

  return 0;
}

void PrelnResidualBiasPluginDynamic::terminate() TRT_NOEXCEPT {
  if (bias_gpu_) {
    hipFree(bias_gpu_);
    bias_gpu_ = nullptr;
  }
  if (scale_gpu_) {
    hipFree(scale_gpu_);
    scale_gpu_ = nullptr;
  }
  if (ele_bias_gpu_) {
    hipFree(ele_bias_gpu_);
    ele_bias_gpu_ = nullptr;
  }
}

nvinfer1::DimsExprs PrelnResidualBiasPluginDynamic::getOutputDimensions(
    int output_index, const nvinfer1::DimsExprs *inputs, int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  if (output_index < 2) {
    return inputs[0];
  } else {  // moving mean and var
    nvinfer1::DimsExprs ret;
    ret.nbDims = 1;
    ret.d[0] = inputs[0].d[2];
    return ret;
  }
}

bool PrelnResidualBiasPluginDynamic::supportsFormatCombination(
    int pos, const nvinfer1::PluginTensorDesc *in_out, int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out, platform::errors::InvalidArgument(
                  "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos, nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos, nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
      PADDLE_THROW(
          platform::errors::Fatal("TRT plugin supported FP16 is not available "
                                  "while with_fp16 is set true."));
#endif
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType PrelnResidualBiasPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

int PrelnResidualBiasPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc, const void *const *inputs,
    void *const *outputs, void *workspace, hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  int hidden = input_dims.d[2];
  const size_t rows = static_cast<size_t>(
      input_dims.d[0] * input_dims.d[1]);  // batch * seq_length
  const size_t cols = static_cast<size_t>(input_dims.d[2]);

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp32";
    const float *input1 = static_cast<const float *>(inputs[0]);
    const float *input2 = static_cast<const float *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train = false;
    const bool is_test = true;
    const uint64_t increment = 0;
    const float epsilon = eps_;
    const float *src = input2;
    const float *residual = input1;
    const float *bias = static_cast<float *>(ele_bias_gpu_);
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data = nullptr;
    float *dst = static_cast<float *>(outputs[1]);
    float *layernorm_dst = static_cast<float *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;
    const int VecSize = 8;
    paddle::operators::FusedLayernormResidualDropoutBiasFunctor<
        float, uint8_t, VecSize, float, false>()(
        rows, cols, seed, dropout_prob, is_upscale_in_train, is_test, increment,
        epsilon, src, residual, bias, scale, layernorm_bias, mask_data, dst,
        layernorm_dst, mean, var, stream);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp16";
    const half *input1 = static_cast<const half *>(inputs[0]);
    const half *input2 = static_cast<const half *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train = false;
    const bool is_test = true;
    const uint64_t increment = 0;
    const float epsilon = eps_;
    const half *src = input2;
    const half *residual = input1;
    const half *bias = static_cast<half *>(ele_bias_gpu_);
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data = nullptr;
    half *dst = static_cast<half *>(outputs[1]);
    half *layernorm_dst = static_cast<half *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;
    const int VecSize = 8;
    paddle::operators::FusedLayernormResidualDropoutBiasFunctor<
        half, uint8_t, VecSize, float, false>()(
        rows, cols, seed, dropout_prob, is_upscale_in_train, is_test, increment,
        epsilon, src, residual, bias, scale, layernorm_bias, mask_data, dst,
        layernorm_dst, mean, var, stream);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) tensorRT plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The PrelnResidualBias TRT Plugin's input type "
                                "should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
