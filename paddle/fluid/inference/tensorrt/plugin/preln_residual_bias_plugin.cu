#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
<<<<<<< HEAD
=======
// Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>

#include "glog/logging.h"
#include "paddle/fluid/inference/tensorrt/plugin/preln_residual_bias_plugin.h"
#include "paddle/fluid/operators/fused/fused_dropout_common.h"
#include "paddle/fluid/operators/fused/fused_layernorm_residual_dropout_bias.h"
#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/fluid/operators/math/bert_encoder_functor.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {
<<<<<<< HEAD
=======
#ifdef TRT_PLUGIN_FP16_AVALIABLE
#define FINAL_MASK 0xffffffff
template <typename T, int NUM>
__inline__ __device__ T warpReduceSumV2(T *val) {
#pragma unroll
  for (int i = 0; i < NUM; i++) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
      val[i] += __shfl_xor_sync(FINAL_MASK, val[i], mask, 32);
  }
  return (T)(0.0f);
}

template <typename T, int NUM>
__inline__ __device__ T blockReduceSumV2(T *val) {
  static __shared__ T shared[NUM][33];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  warpReduceSumV2<T, NUM>(val);

  if (lane == 0) {
#pragma unroll
    for (int i = 0; i < NUM; i++) {
      shared[i][wid] = val[i];
    }
  }
  __syncthreads();

  bool is_mask = threadIdx.x < (blockDim.x / 32.f);
#pragma unroll
  for (int i = 0; i < NUM; i++) {
    val[i] = is_mask ? shared[i][lane] : (T)(0.0f);
  }
  warpReduceSumV2<T, NUM>(val);
  return (T)0.0f;
}

template <int UNROLL_FACTOR>
__global__ void generalAddBiasResidualLayerNormOpt2(
    half2 *normed_output,
    half2 *output,
    const half2 *__restrict bias,
    const half2 *__restrict src,
    const half2 *__restrict residual,
    const half2 *__restrict gamma,
    const half2 *__restrict beta,
    int m,
    int n,
    float epsilon) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  __shared__ float s_mean;
  __shared__ float s_variance;
  float x_sum = 0.0f;
  float x2_sum = 0.0f;
  const int b_offset = blockIdx.x * n;

#pragma unroll UNROLL_FACTOR
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int index = b_offset + i;
    float val_1 = 0.0f;
    float val_2 = 0.0f;
    half2 tmp;

    if (bias) {
      tmp = __ldg(&bias[i]);
      val_1 += static_cast<float>(tmp.x);
      val_2 += static_cast<float>(tmp.y);
    }
    {
      tmp = __ldg(&residual[index]);
      val_1 += static_cast<float>(tmp.x);
      val_2 += static_cast<float>(tmp.y);
    }
    {
      tmp = __ldg(&src[index]);
      val_1 += static_cast<float>(tmp.x);
      val_2 += static_cast<float>(tmp.y);
    }
    tmp.x = __float2half_rn(val_1);
    tmp.y = __float2half_rn(val_2);
    output[index] = tmp;
    x_sum += val_1 + val_2;
    x2_sum += val_1 * val_1 + val_2 * val_2;
  }
  float sums[2];
  sums[0] = x_sum;
  sums[1] = x2_sum;
  blockReduceSumV2<float, 2>(sums);

  if (threadIdx.x == 0) {
    s_mean = sums[0] / n / 2;
    s_variance = rsqrtf(sums[1] / n / 2 - s_mean * s_mean + epsilon);
  }
  __syncthreads();

  half2 mean_2 = __float2half2_rn(s_mean);
  half2 var_2 = __float2half2_rn(s_variance);

#pragma unroll UNROLL_FACTOR
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int index = b_offset + i;
    half2 val = __hmul2(__hmul2(__hsub2(output[index], mean_2), var_2),
                        __ldg(&gamma[i]));
    if (beta) {
      val = __hadd2(val, __ldg(&beta[i]));
    }
    normed_output[index] = val;
  }
#endif
}

#define HALF2_ADD_BIAS_RESIDUAL_LAYERNORM_OPT2(UNROLL_FACTOR)                \
  generalAddBiasResidualLayerNormOpt2<UNROLL_FACTOR>                         \
      <<<rows, block, 0, stream>>>(reinterpret_cast<half2 *>(layernorm_dst), \
                                   reinterpret_cast<half2 *>(dst),           \
                                   (const half2 *)bias,                      \
                                   (const half2 *)input2,                    \
                                   (const half2 *)input1,                    \
                                   (const half2 *)fp16_scale_gpu_,           \
                                   (const half2 *)fp16_bias_gpu_,            \
                                   rows,                                     \
                                   half_n,                                   \
                                   epsilon);

#endif

>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
using half = phi::dtype::float16;

#if IS_TRT_VERSION_GE(6000)
int PrelnResidualBiasPluginDynamic::initialize() TRT_NOEXCEPT {
  hipMalloc(&bias_gpu_, sizeof(float) * bias_size_);
  hipMemcpy(bias_gpu_,
             bias_.data(),
             bias_size_ * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&scale_gpu_, sizeof(float) * scale_size_);
  hipMemcpy(scale_gpu_,
             scale_.data(),
             scale_size_ * sizeof(float),
             hipMemcpyHostToDevice);
<<<<<<< HEAD

  if (with_fp16_) {
    hipMalloc(&ele_bias_gpu_, sizeof(half) * ele_bias_size_);
    hipMemcpy(ele_bias_gpu_,
               fp16_ele_bias_.data(),
               ele_bias_size_ * sizeof(half),
               hipMemcpyHostToDevice);
  } else {
    hipMalloc(&ele_bias_gpu_, sizeof(float) * ele_bias_size_);
    hipMemcpy(ele_bias_gpu_,
               fp32_ele_bias_.data(),
               ele_bias_size_ * sizeof(float),
               hipMemcpyHostToDevice);
  }
=======
  if (with_fp16_) {
    hipMalloc(&fp16_bias_gpu_, sizeof(half) * bias_size_);
    hipMemcpy(fp16_bias_gpu_,
               fp16_bias_.data(),
               bias_size_ * sizeof(half),
               hipMemcpyHostToDevice);
    hipMalloc(&fp16_scale_gpu_, sizeof(half) * scale_size_);
    hipMemcpy(fp16_scale_gpu_,
               fp16_scale_.data(),
               scale_size_ * sizeof(half),
               hipMemcpyHostToDevice);
  }
  if (ele_bias_size_ > 0) {
    if (with_fp16_) {
      hipMalloc(&ele_bias_gpu_, sizeof(half) * ele_bias_size_);
      hipMemcpy(ele_bias_gpu_,
                 fp16_ele_bias_.data(),
                 ele_bias_size_ * sizeof(half),
                 hipMemcpyHostToDevice);
    } else {
      hipMalloc(&ele_bias_gpu_, sizeof(float) * ele_bias_size_);
      hipMemcpy(ele_bias_gpu_,
                 fp32_ele_bias_.data(),
                 ele_bias_size_ * sizeof(float),
                 hipMemcpyHostToDevice);
    }
  } else {
    ele_bias_gpu_ = nullptr;
  }
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f

  return 0;
}

void PrelnResidualBiasPluginDynamic::terminate() TRT_NOEXCEPT {
  if (bias_gpu_) {
    hipFree(bias_gpu_);
    bias_gpu_ = nullptr;
  }
<<<<<<< HEAD
=======
  if (fp16_bias_gpu_) {
    hipFree(fp16_bias_gpu_);
    fp16_bias_gpu_ = nullptr;
  }
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  if (scale_gpu_) {
    hipFree(scale_gpu_);
    scale_gpu_ = nullptr;
  }
<<<<<<< HEAD
=======
  if (fp16_scale_gpu_) {
    hipFree(fp16_scale_gpu_);
    fp16_scale_gpu_ = nullptr;
  }
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  if (ele_bias_gpu_) {
    hipFree(ele_bias_gpu_);
    ele_bias_gpu_ = nullptr;
  }
}

nvinfer1::IPluginV2DynamicExt *PrelnResidualBiasPluginDynamic::clone() const
    TRT_NOEXCEPT {
  PrelnResidualBiasPluginDynamic *ptr = nullptr;
  if (with_fp16_) {
    ptr = new PrelnResidualBiasPluginDynamic(bias_.data(),
                                             scale_.data(),
                                             fp16_ele_bias_.data(),
                                             bias_size_,
                                             scale_size_,
                                             ele_bias_size_,
                                             eps_,
                                             with_fp16_);
  } else {
    ptr = new PrelnResidualBiasPluginDynamic(bias_.data(),
                                             scale_.data(),
                                             fp32_ele_bias_.data(),
                                             bias_size_,
                                             scale_size_,
                                             ele_bias_size_,
                                             eps_,
                                             with_fp16_);
  }

  ptr->bias_gpu_ = bias_gpu_;
<<<<<<< HEAD
  ptr->scale_gpu_ = scale_gpu_;
=======
  ptr->fp16_bias_gpu_ = fp16_bias_gpu_;
  ptr->scale_gpu_ = scale_gpu_;
  ptr->fp16_scale_gpu_ = fp16_scale_gpu_;
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  ptr->ele_bias_gpu_ = ele_bias_gpu_;
  return ptr;
}

const char *PrelnResidualBiasPluginDynamic::getPluginType() const TRT_NOEXCEPT {
  return "preln_residual_bias_plugin_dynamic";
}

int PrelnResidualBiasPluginDynamic::getNbOutputs() const TRT_NOEXCEPT {
  return 2;
}

size_t PrelnResidualBiasPluginDynamic::getSerializationSize() const
    TRT_NOEXCEPT {
<<<<<<< HEAD
  size_t ser_size = SerializedSize(bias_) + SerializedSize(scale_) +
=======
  size_t ser_size = SerializedSize(bias_) + SerializedSize(fp16_bias_) +
                    SerializedSize(scale_) + SerializedSize(fp16_scale_) +
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
                    SerializedSize(fp32_ele_bias_) +
                    SerializedSize(fp16_ele_bias_) +
                    SerializedSize(bias_size_) + SerializedSize(scale_size_) +
                    SerializedSize(ele_bias_size_) + SerializedSize(eps_) +
                    SerializedSize(with_fp16_);
  return ser_size;
}
void PrelnResidualBiasPluginDynamic::serialize(void *buffer) const
    TRT_NOEXCEPT {
  SerializeValue(&buffer, bias_);
<<<<<<< HEAD
  SerializeValue(&buffer, scale_);
=======
  SerializeValue(&buffer, fp16_bias_);
  SerializeValue(&buffer, scale_);
  SerializeValue(&buffer, fp16_scale_);
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
  SerializeValue(&buffer, fp32_ele_bias_);
  SerializeValue(&buffer, fp16_ele_bias_);
  SerializeValue(&buffer, bias_size_);
  SerializeValue(&buffer, scale_size_);
  SerializeValue(&buffer, ele_bias_size_);
  SerializeValue(&buffer, eps_);
  SerializeValue(&buffer, with_fp16_);
}

nvinfer1::DimsExprs PrelnResidualBiasPluginDynamic::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  if (output_index < 2) {
    return inputs[0];
  } else {  // moving mean and var
    nvinfer1::DimsExprs ret;
    ret.nbDims = 1;
    ret.d[0] = inputs[0].d[2];
    return ret;
  }
}

bool PrelnResidualBiasPluginDynamic::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
      return (in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
      PADDLE_THROW(
          platform::errors::Fatal("TRT plugin supported FP16 is not available "
                                  "while with_fp16 is set true."));
#endif
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

void PrelnResidualBiasPluginDynamic::configurePlugin(
    const nvinfer1::DynamicPluginTensorDesc *in,
    int nb_inputs,
    const nvinfer1::DynamicPluginTensorDesc *out,
    int nb_outputs) TRT_NOEXCEPT {}

size_t PrelnResidualBiasPluginDynamic::getWorkspaceSize(
    const nvinfer1::PluginTensorDesc *inputs,
    int nb_inputs,
    const nvinfer1::PluginTensorDesc *outputs,
    int nb_outputs) const TRT_NOEXCEPT {
  return 0;
}

nvinfer1::DataType PrelnResidualBiasPluginDynamic::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  return input_types[0];
}

void PrelnResidualBiasPluginDynamic::destroy() TRT_NOEXCEPT { delete this; }

int PrelnResidualBiasPluginDynamic::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  int hidden = input_dims.d[2];
  const size_t rows = static_cast<size_t>(
      input_dims.d[0] * input_dims.d[1]);  // batch * seq_length
  const size_t cols = static_cast<size_t>(input_dims.d[2]);

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp32";
    const float *input1 = static_cast<const float *>(inputs[0]);
    const float *input2 = static_cast<const float *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train = false;
    const bool is_test = true;
    const uint64_t increment = 0;
    const float epsilon = eps_;
    const float *src = input2;
    const float *residual = input1;
    const float *bias = static_cast<float *>(ele_bias_gpu_);
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data = nullptr;
    float *dst = static_cast<float *>(outputs[1]);
    float *layernorm_dst = static_cast<float *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;
    const int VecSize = 8;
    paddle::operators::FusedLayernormResidualDropoutBiasFunctor<float,
                                                                uint8_t,
                                                                VecSize,
                                                                float,
                                                                false>()(
        rows,
        cols,
        seed,
        dropout_prob,
        is_upscale_in_train,
        is_test,
        increment,
        epsilon,
        src,
        residual,
        bias,
        scale,
        layernorm_bias,
        mask_data,
        dst,
        layernorm_dst,
        mean,
        var,
        stream);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. PrelnResidualBias-->fp16";
    const half *input1 = static_cast<const half *>(inputs[0]);
    const half *input2 = static_cast<const half *>(inputs[1]);

    uint64_t seed = 0;
    const float dropout_prob = 0.;
    const bool is_upscale_in_train = false;
    const bool is_test = true;
    const uint64_t increment = 0;
    const float epsilon = eps_;
    const half *src = input2;
    const half *residual = input1;
    const half *bias = static_cast<half *>(ele_bias_gpu_);
    const float *scale = scale_gpu_;
    const float *layernorm_bias = bias_gpu_;
    uint8_t *mask_data = nullptr;
    half *dst = static_cast<half *>(outputs[1]);
    half *layernorm_dst = static_cast<half *>(outputs[0]);
    float *mean = nullptr;
    float *var = nullptr;
    const int VecSize = 8;
<<<<<<< HEAD
=======
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
    // if hidden is even, use half2 kernel generalAddBiasResidualLayerNormOpt2
    if (hidden % 2 == 0) {
      int half_n = hidden / 2;
      int half_n_32 = (half_n + 31) / 32 * 32;
      dim3 block(std::min(half_n_32, 512));
      int rolls_per_thread = half_n / block.x;
      int unroll_factor = 8;
      while (unroll_factor > rolls_per_thread && unroll_factor > 1) {
        unroll_factor /= 2;
      }
      switch (unroll_factor) {
        case 1:
          HALF2_ADD_BIAS_RESIDUAL_LAYERNORM_OPT2(1);
          break;
        case 2:
          HALF2_ADD_BIAS_RESIDUAL_LAYERNORM_OPT2(2);
          break;
        case 4:
          HALF2_ADD_BIAS_RESIDUAL_LAYERNORM_OPT2(4);
          break;
        case 8:
          HALF2_ADD_BIAS_RESIDUAL_LAYERNORM_OPT2(8);
          break;
        default:
          PADDLE_THROW(platform::errors::Fatal(
              "Invalid UNROLL_FACTOR in preln_residual_bias trt plugin."));
      }
    } else {
      paddle::operators::FusedLayernormResidualDropoutBiasFunctor<half,
                                                                  uint8_t,
                                                                  VecSize,
                                                                  float,
                                                                  false>()(
          rows,
          cols,
          seed,
          dropout_prob,
          is_upscale_in_train,
          is_test,
          increment,
          epsilon,
          src,
          residual,
          bias,
          scale,
          layernorm_bias,
          mask_data,
          dst,
          layernorm_dst,
          mean,
          var,
          stream);
    }
#else
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
    paddle::operators::FusedLayernormResidualDropoutBiasFunctor<half,
                                                                uint8_t,
                                                                VecSize,
                                                                float,
                                                                false>()(
        rows,
        cols,
        seed,
        dropout_prob,
        is_upscale_in_train,
        is_test,
        increment,
        epsilon,
        src,
        residual,
        bias,
        scale,
        layernorm_bias,
        mask_data,
        dst,
        layernorm_dst,
        mean,
        var,
        stream);
<<<<<<< HEAD
=======
#endif
>>>>>>> 43b92b633f5d2db98f45d4b9597e5389f6f9712f
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) tensorRT plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(
        platform::errors::Fatal("The PrelnResidualBias TRT Plugin's input type "
                                "should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}

const char *PrelnResidualBiasPluginDynamicCreator::getPluginName() const
    TRT_NOEXCEPT {
  return "preln_residual_bias_plugin_dynamic";
}

const char *PrelnResidualBiasPluginDynamicCreator::getPluginVersion() const
    TRT_NOEXCEPT {
  return "1";
}

nvinfer1::IPluginV2 *PrelnResidualBiasPluginDynamicCreator::deserializePlugin(
    const char *name,
    const void *serial_data,
    size_t serial_length) TRT_NOEXCEPT {
  return new PrelnResidualBiasPluginDynamic(serial_data, serial_length);
}

#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
