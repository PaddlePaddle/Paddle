#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thread>  // NOLINT
#include <vector>

#include "gtest/gtest.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/platform/device_context.h"

namespace paddle {
namespace memory {

const int NUM_STREAMS = 8;
const int N = 2;
const float DELTA = 1e-1;

using CudaDevCtxVec = std::vector<std::unique_ptr<platform::CUDADeviceContext>>;

__global__ void kernel(float *x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = 3.14159 * i;
  }
}

void CheckKernelOutput(float *x, int n) {
  auto host_x = std::unique_ptr<float[]>(new float[n]);
  for (int i = 0; i < n; ++i) {
    EXPECT_TRUE(hipSuccess == hipMemcpy(host_x.get(), x, n * sizeof(float),
                                          hipMemcpyDeviceToHost));
    EXPECT_GE(host_x[i] + DELTA, 3.14159f * i);
    EXPECT_LE(host_x[i] - DELTA, 3.14159f * i);
  }
}

void MultiStreamCompute(float **data, float **second_data,
                        const platform::CUDADeviceContext &ctx) {
  // multi-streams
  AllocationPtr allocation_ptr = Alloc(ctx, N * sizeof(float));
  EXPECT_GE(allocation_ptr->size(), N * sizeof(float));
  *data = reinterpret_cast<float *>(allocation_ptr->ptr());
  kernel<<<1, 64, 0, ctx.stream()>>>(*data, N);

  // allocate and compute on same stream again
  allocation_ptr = Alloc(ctx, N * sizeof(float));
  EXPECT_GE(allocation_ptr->size(), N * sizeof(float));
  *second_data = reinterpret_cast<float *>(allocation_ptr->ptr());
  kernel<<<1, 64, 0, ctx.stream()>>>(*second_data, N);
}

TEST(Malloc, CUDADeviceContextMultiStream) {
  auto place = platform::CUDAPlace(0);
  platform::SetDeviceId(0);

  AllocationPtr main_stream_alloc_ptr = Alloc(place, N * sizeof(float));
  EXPECT_GE(main_stream_alloc_ptr->size(), N * sizeof(float));
  float *main_stream_data =
      reinterpret_cast<float *>(main_stream_alloc_ptr->ptr());

  float *data[NUM_STREAMS];
  float *second_data[NUM_STREAMS];
  CudaDevCtxVec dev_ctx;

  // default stream
  kernel<<<1, 64>>>(main_stream_data, N);
  main_stream_alloc_ptr.reset();

  for (int i = 0; i < NUM_STREAMS; ++i) {
    dev_ctx.push_back(std::unique_ptr<platform::CUDADeviceContext>(
        new platform::CUDADeviceContext(place)));
    MultiStreamCompute(&data[i], &second_data[i], *dev_ctx[i]);
  }

  EXPECT_TRUE(hipSuccess == hipDeviceSynchronize());
  for (int i = 0; i < NUM_STREAMS; ++i) {
    CheckKernelOutput(data[i], N);
    CheckKernelOutput(second_data[i], N);
  }
}

TEST(Malloc, CUDADeviceContextMultiThreadMultiStream) {
  auto place = platform::CUDAPlace(0);
  platform::SetDeviceId(0);

  AllocationPtr main_stream_alloc_ptr = Alloc(place, N * sizeof(float));
  EXPECT_GE(main_stream_alloc_ptr->size(), N * sizeof(float));
  float *main_stream_data =
      reinterpret_cast<float *>(main_stream_alloc_ptr->ptr());

  float *data[NUM_STREAMS];
  float *second_data[NUM_STREAMS];
  CudaDevCtxVec dev_ctx;
  std::vector<std::thread> threads;

  // default stream
  kernel<<<1, 64>>>(main_stream_data, N);
  main_stream_alloc_ptr.reset();

  for (int i = 0; i < NUM_STREAMS; ++i) {
    dev_ctx.push_back(std::unique_ptr<platform::CUDADeviceContext>(
        new platform::CUDADeviceContext(place)));
    threads.push_back(std::thread(MultiStreamCompute, &data[i], &second_data[i],
                                  std::cref(*dev_ctx[i])));
  }

  for (int i = 0; i < NUM_STREAMS; ++i) {
    threads[i].join();
  }

  EXPECT_TRUE(hipSuccess == hipDeviceSynchronize());
  for (int i = 0; i < NUM_STREAMS; ++i) {
    CheckKernelOutput(data[i], N);
    CheckKernelOutput(second_data[i], N);
  }
}

TEST(Malloc, AllocZero) {
  auto place = platform::CUDAPlace(0);
  AllocationPtr allocation_ptr = Alloc(place, 0);
  EXPECT_GE(allocation_ptr->size(), 0);
}
}  // namespace memory
}  // namespace paddle
