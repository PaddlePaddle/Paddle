#include "hip/hip_runtime.h"
// Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gtest/gtest.h"
#include "paddle/fluid/memory/allocation/cuda_device_context_allocator.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/platform/device_context.h"

namespace paddle {
namespace memory {

const int NUM_STREAMS = 8;
const int N = 1 << 10;
const float DELTA = 1e-1;

__global__ void kernel(float *x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = 3.14159 * i;
  }
}

void CheckKernelOutput(float *x, int n) {
  float *host_x = new float[n];
  for (int i = 0; i < n; ++i) {
    EXPECT_TRUE(hipSuccess == hipMemcpy(host_x, x, n * sizeof(float),
                                          hipMemcpyDeviceToHost));
    EXPECT_GE(host_x[i] + DELTA, 3.14159f * i);
    EXPECT_LE(host_x[i] - DELTA, 3.14159f * i);
  }
  delete[] host_x;
}

TEST(Malloc, CUDADeviceContextMultiStream) {
  auto place = platform::CUDAPlace(0);

  CUDADeviceContextAllocator allocator(place);
  EXPECT_TRUE(hipSuccess == hipSetDevice(0));

  AllocationPtr main_stream_alloc_ptr = Alloc(place, N * sizeof(float));
  EXPECT_EQ(main_stream_alloc_ptr->size(), N * sizeof(float));
  float *main_stream_data =
      reinterpret_cast<float *>(main_stream_alloc_ptr->ptr());

  hipStream_t streams[NUM_STREAMS];
  float *data[NUM_STREAMS];

  for (int i = 0; i < NUM_STREAMS; ++i) {
    // default stream
    kernel<<<1, 64>>>(main_stream_data, N);

    paddle::platform::CUDADeviceContext dev_ctx(place);
    AllocationPtr allocation_ptr = Alloc(dev_ctx, N * sizeof(float));
    VLOG(4) << "Get ptr";
    EXPECT_EQ(allocation_ptr->size(), N * sizeof(float));
    data[i] = reinterpret_cast<float *>(allocation_ptr->ptr());

    // multi-streams
    streams[i] = dev_ctx.stream();
    kernel<<<1, 64, 0, streams[i]>>>(data[i], N);
  }

  EXPECT_TRUE(hipSuccess == hipDeviceSynchronize());
  CheckKernelOutput(main_stream_data, N);
  for (int i = 0; i < NUM_STREAMS; ++i) {
    hipStreamSynchronize(streams[i]);
    CheckKernelOutput(data[i], N);
  }

  EXPECT_TRUE(hipSuccess == hipDeviceReset());
}

}  // namespace memory
}  // namespace paddle
