#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>
#endif

#include <thread>  // NOLINT
#include <vector>

#include "gtest/gtest.h"
#include "paddle/fluid/memory/allocation/allocator_facade.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/platform/cuda_graph_with_memory_pool.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/core/stream.h"

namespace paddle {
namespace memory {

__global__ void add_kernel(int *x, int n) {
  int thread_num = gridDim.x * blockDim.x;
  int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = thread_id; i < n; i += thread_num) {
    atomicAdd(x + i, thread_id);
  }
}

void CheckMemLeak(const platform::CUDAPlace &place) {
  uint64_t cuda_malloc_size =
      platform::RecordedGpuMallocSize(place.GetDeviceId());
  ASSERT_EQ(cuda_malloc_size, 0) << "Found " << cuda_malloc_size
                                 << " bytes memory that not released yet,"
                                 << " there may be a memory leak problem";
}

class StreamSafeCUDAAllocTest : public ::testing::Test {
 protected:
  void SetUp() override {
    place_ = platform::CUDAPlace();
    stream_num_ = 64;
    grid_num_ = 1;
    block_num_ = 32;
    data_num_ = 131072;
    workspace_size_ = data_num_ * sizeof(int);

    // alloc workspace for each stream
    for (size_t i = 0; i < stream_num_; ++i) {
      gpuStream_t stream;
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream));
#endif

      std::shared_ptr<Allocation> allocation =
          AllocShared(place_, workspace_size_,
                      phi::Stream(reinterpret_cast<phi::StreamId>(stream)));
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemset(allocation->ptr(), 0, allocation->size()));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemset(allocation->ptr(), 0, allocation->size()));
#endif

      streams_.emplace_back(stream);
      workspaces_.emplace_back(allocation);
    }

    result_ = Alloc(place_, stream_num_ * workspace_size_);
  }

  void SingleStreamRun(size_t idx) {
    // for all stream i,
    // stream idx lauch a kernel to add (j % thread_num) to workspaces_[i][j]
    for (size_t i = 0; i < stream_num_; ++i) {
      int *x = reinterpret_cast<int *>(workspaces_[i]->ptr());
      add_kernel<<<grid_num_, block_num_, 0, streams_[idx]>>>(x, data_num_);
      RecordStream(workspaces_[i], streams_[idx]);
    }
  }

  void CopyResultAsync() {
    for (size_t i = 0; i < stream_num_; ++i) {
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(
          reinterpret_cast<int *>(result_->ptr()) + i * data_num_,
          workspaces_[i]->ptr(), workspace_size_, hipMemcpyDeviceToDevice));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(
          reinterpret_cast<int *>(result_->ptr()) + i * data_num_,
          workspaces_[i]->ptr(), workspace_size_, hipMemcpyDeviceToDevice));
#endif
    }
  }

  void MultiStreamRun() {
    for (size_t i = 0; i < stream_num_; ++i) {
      SingleStreamRun(i);
    }
    CopyResultAsync();
    workspaces_.clear();  // fast_gc
    hipDeviceSynchronize();
  }

  void MultiThreadMUltiStreamRun() {
    std::vector<std::thread> threads;
    for (size_t i = 0; i < stream_num_; ++i) {
      threads.push_back(
          std::thread(&StreamSafeCUDAAllocTest::SingleStreamRun, this, i));
    }
    for (size_t i = 0; i < stream_num_; ++i) {
      threads[i].join();
    }
    CopyResultAsync();
    workspaces_.clear();  // fast_gc
    hipDeviceSynchronize();
  }

  void CheckResult() {
    auto result_host = std::unique_ptr<int[]>(new int[result_->size()]);
#ifdef PADDLE_WITH_CUDA
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(result_host.get(), result_->ptr(),
                                          result_->size(),
                                          hipMemcpyDeviceToHost));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(result_host.get(), result_->ptr(),
                                         result_->size(),
                                         hipMemcpyDeviceToHost));
#endif
    size_t thread_num = grid_num_ * block_num_;
    for (size_t i = 0; i < stream_num_; ++i) {
      for (size_t j = 0; j < data_num_; ++j) {
        EXPECT_TRUE(result_host[i * stream_num_ + j] ==
                    (j % thread_num) * stream_num_);
      }
    }
    result_.reset();
  }

  void TearDown() override {
#ifdef PADDLE_WITH_CUDA
    PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#else
    PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif
    for (gpuStream_t stream : streams_) {
      Release(place_, stream);
    }

    for (size_t i = 1; i < stream_num_; ++i) {
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(streams_[i]));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(streams_[i]));
#endif
    }

    CheckMemLeak(place_);
  }

  size_t stream_num_;
  size_t grid_num_;
  size_t block_num_;
  size_t data_num_;
  size_t workspace_size_;
  platform::CUDAPlace place_;
  std::vector<gpuStream_t> streams_;
  std::vector<std::shared_ptr<Allocation>> workspaces_;
  allocation::AllocationPtr result_;
};

TEST_F(StreamSafeCUDAAllocTest, CUDAMutilStreamTest) {
  MultiStreamRun();
  CheckResult();
}

TEST_F(StreamSafeCUDAAllocTest, CUDAMutilThreadMutilStreamTest) {
  MultiThreadMUltiStreamRun();
  CheckResult();
}

TEST(StreamSafeCUDAAllocInterfaceTest, AllocInterfaceTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  size_t alloc_size = 256;

  std::shared_ptr<Allocation> allocation_implicit_stream =
      AllocShared(place, alloc_size);
  EXPECT_GE(allocation_implicit_stream->size(), alloc_size);

  void *address = allocation_implicit_stream->ptr();
  allocation_implicit_stream.reset();

  gpuStream_t default_stream =
      dynamic_cast<platform::CUDADeviceContext *>(
          paddle::platform::DeviceContextPool::Instance().Get(place))
          ->stream();
  allocation::AllocationPtr allocation_unique =
      Alloc(place, alloc_size, default_stream);
  EXPECT_GE(allocation_unique->size(), alloc_size);
  EXPECT_EQ(allocation_unique->ptr(), address);
  allocation_unique.reset();

  Release(place);
  CheckMemLeak(place);
}

TEST(StreamSafeCUDAAllocInterfaceTest, GetAllocatorInterfaceTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  size_t alloc_size = 256;

  allocation::AllocationPtr allocation_implicit_stream =
      Alloc(place, alloc_size);
  EXPECT_GE(allocation_implicit_stream->size(), alloc_size);
  void *address = allocation_implicit_stream->ptr();
  allocation_implicit_stream.reset();

  auto &instance = allocation::AllocatorFacade::Instance();
  const std::shared_ptr<Allocator> &allocator = instance.GetAllocator(place);

  allocation::AllocationPtr allocation_from_allocator =
      allocator->Allocate(alloc_size);
  EXPECT_GE(allocation_from_allocator->size(), alloc_size);
  EXPECT_EQ(allocation_from_allocator->ptr(), address);
  allocation_from_allocator.reset();

  Release(place);
  CheckMemLeak(place);
}

TEST(StreamSafeCUDAAllocInterfaceTest, ZeroSizeRecordStreamTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  std::shared_ptr<Allocation> zero_size_allocation = AllocShared(place, 0);
  EXPECT_EQ(zero_size_allocation->ptr(), nullptr);

  gpuStream_t stream;
#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream));
#endif

  EXPECT_NO_THROW(RecordStream(zero_size_allocation, stream));

#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(stream));
#endif
}

TEST(StreamSafeCUDAAllocInterfaceTest, GetStreamInterfaceTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  size_t alloc_size = 256;

  gpuStream_t default_stream =
      dynamic_cast<platform::CUDADeviceContext *>(
          paddle::platform::DeviceContextPool::Instance().Get(place))
          ->stream();
  std::shared_ptr<Allocation> allocation_implicit_stream =
      AllocShared(place, alloc_size);
  EXPECT_EQ(GetStream(allocation_implicit_stream), default_stream);

  gpuStream_t new_stream;
#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&new_stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&new_stream));
#endif

  std::shared_ptr<Allocation> allocation_new_stream =
      AllocShared(place, alloc_size,
                  phi::Stream(reinterpret_cast<phi::StreamId>(new_stream)));
  EXPECT_EQ(GetStream(allocation_new_stream), new_stream);

#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(new_stream));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(new_stream));
#endif

  allocation_implicit_stream.reset();
  allocation_new_stream.reset();
  Release(place);
  CheckMemLeak(place);
}

#ifdef PADDLE_WITH_CUDA
TEST(StreamSafeCUDAAllocInterfaceTest, CUDAGraphExceptionTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  size_t alloc_size = 1;
  std::shared_ptr<Allocation> allocation = AllocShared(place, alloc_size);

  platform::BeginCUDAGraphCapture(place, hipStreamCaptureModeGlobal);
  EXPECT_THROW(AllocShared(place, alloc_size), paddle::platform::EnforceNotMet);
  EXPECT_THROW(Alloc(place, alloc_size), paddle::platform::EnforceNotMet);
  EXPECT_THROW(Release(place), paddle::platform::EnforceNotMet);
  EXPECT_THROW(allocation::AllocatorFacade::Instance().GetAllocator(place),
               paddle::platform::EnforceNotMet);
  EXPECT_THROW(
      AllocShared(place, alloc_size,
                  phi::Stream(reinterpret_cast<phi::StreamId>(nullptr))),
      paddle::platform::EnforceNotMet);
  EXPECT_THROW(Alloc(place, alloc_size, nullptr),
               paddle::platform::EnforceNotMet);
  EXPECT_THROW(Release(place, nullptr), paddle::platform::EnforceNotMet);
  EXPECT_THROW(RecordStream(allocation, nullptr),
               paddle::platform::EnforceNotMet);
  EXPECT_THROW(GetStream(allocation), paddle::platform::EnforceNotMet);
  platform::EndCUDAGraphCapture();

  allocation.reset();
  Release(place);
  CheckMemLeak(place);
}
#endif

TEST(StreamSafeCUDAAllocRetryTest, RetryTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  gpuStream_t stream1, stream2;
#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream1));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream2));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream1));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream2));
#endif
  size_t available_size = platform::GpuAvailableMemToAlloc();
  // alloc_size < available_size < 2 * alloc_size,
  // so the second alloc will fail and retry
  size_t alloc_size = available_size / 4 * 3;

  allocation::AllocationPtr allocation1 = Alloc(place, alloc_size, stream1);
  allocation::AllocationPtr allocation2;

  std::thread th([&allocation2, &place, &stream2, alloc_size]() {
    std::this_thread::sleep_for(std::chrono::seconds(1));
    allocation2 = Alloc(place, alloc_size, stream2);
  });
  allocation1.reset();  // free but not release
  th.join();
  EXPECT_GE(allocation2->size(), alloc_size);
  allocation2.reset();

#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif

  Release(place, stream1);
  Release(place, stream2);
  CheckMemLeak(place);
}

}  // namespace memory
}  // namespace paddle
