#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>
#endif

#include <thread>  // NOLINT
#include <vector>

#include "gtest/gtest.h"
#include "paddle/fluid/memory/allocation/allocator_facade.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"

namespace paddle {
namespace memory {

__global__ void add_kernel(int *x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    atomicAdd(x + i, tid);
  }
}

void CheckMemLeak(const platform::CUDAPlace &place) {
  uint64_t cuda_malloc_size =
      platform::RecordedGpuMallocSize(place.GetDeviceId());
  ASSERT_EQ(cuda_malloc_size, 0) << "Found " << cuda_malloc_size
                                 << " bytes memory that not released yet,"
                                 << " there may be a memory leak problem";
}

class StreamSafeCUDAAllocTest : public ::testing::Test {
 protected:
  void SetUp() override {
    place_ = platform::CUDAPlace();
    stream_num_ = 64;
    grid_num_ = 1;
    block_num_ = 64;
    data_num_ = 64;
    default_stream = nullptr;

    streams_.reserve(stream_num_);
    streams_.emplace_back(default_stream);
    for (size_t i = 1; i < stream_num_; ++i) {
      gpuStream_t stream;
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream));
#endif
      streams_.emplace_back(stream);
    }

    for (size_t i = 0; i < stream_num_; ++i) {
      size_t allocation_size = data_num_ * sizeof(int);
      std::shared_ptr<Allocation> allocation =
          AllocShared(place_, allocation_size, streams_[i]);
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemset(allocation->ptr(), 0, allocation->size()));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemset(allocation->ptr(), 0, allocation->size()));
#endif
      allocations_.emplace_back(allocation);
    }
  }

  void SingleStreamRun(size_t idx) {
    for (size_t i = 0; i < stream_num_; ++i) {
      int *x = reinterpret_cast<int *>(allocations_[i]->ptr());
      add_kernel<<<grid_num_, block_num_, 0, streams_[idx]>>>(x, data_num_);
      if (i != idx) {
        RecordStream(allocations_[i].get(), streams_[idx]);
      }
    }
  }

  void MultiStreamRun() {
    for (int i = 0; i < stream_num_; ++i) {
      SingleStreamRun(i);
    }
    allocations_.clear();  // fast_gc
  }

  void MultiThreadMUltiStreamRun() {
    std::vector<std::thread> threads;
    for (size_t i = 0; i < stream_num_; ++i) {
      threads.push_back(
          std::thread(&StreamSafeCUDAAllocTest::SingleStreamRun, this, i));
    }
    for (size_t i = 0; i < stream_num_; ++i) {
      threads[i].join();
    }
    allocations_.clear();  // fast_gc
  }

  void CheckResult() {
    auto host_x = std::unique_ptr<int[]>(new int[data_num_]);
    size_t thread_num = grid_num_ * block_num_;
    for (int i = 0; i < stream_num_; ++i) {
// tricky code, the allocations are still accessible even though
// allocations_.clear() has been called
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemcpy(host_x.get(), allocations_[i]->ptr(),
                     data_num_ * sizeof(int), hipMemcpyDeviceToHost));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpy(host_x.get(), allocations_[i]->ptr(),
                                           data_num_ * sizeof(int),
                                           hipMemcpyDeviceToHost));
#endif
      for (int j = 0; j < data_num_; ++j) {
        EXPECT_TRUE(host_x[j] == (j % thread_num) * stream_num_);
      }
    }
  }

  void TearDown() override {
#ifdef PADDLE_WITH_CUDA
    PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#else
    PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif
    for (gpuStream_t stream : streams_) {
      Release(place_, stream);
    }

    for (size_t i = 1; i < stream_num_; ++i) {
#ifdef PADDLE_WITH_CUDA
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(streams_[i]));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipStreamDestroy(streams_[i]));
#endif
    }

    CheckMemLeak(place_);
  }

  size_t stream_num_;
  size_t grid_num_;
  size_t block_num_;
  size_t data_num_;
  platform::CUDAPlace place_;
  gpuStream_t default_stream;
  std::vector<gpuStream_t> streams_;
  std::vector<std::shared_ptr<Allocation>> allocations_;
};

TEST_F(StreamSafeCUDAAllocTest, CUDAMutilStreamTest) {
  MultiStreamRun();
  CheckResult();
}

TEST_F(StreamSafeCUDAAllocTest, CUDAMutilThreadMutilStreamTest) {
  MultiThreadMUltiStreamRun();
  CheckResult();
}

TEST(StreamSafeCUDAAllocInterfaceTest, AllocInterfaceTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  size_t alloc_size = 256;

  std::shared_ptr<Allocation> allocation_implicit_stream =
      AllocShared(place, alloc_size);
  EXPECT_GE(allocation_implicit_stream->size(), alloc_size);

  void *address = allocation_implicit_stream->ptr();
  allocation_implicit_stream.reset();

  gpuStream_t default_stream = nullptr;
  allocation::AllocationPtr allocation_unique =
      Alloc(place, alloc_size, default_stream);
  EXPECT_GE(allocation_unique->size(), alloc_size);
  EXPECT_EQ(allocation_unique->ptr(), address);
  allocation_unique.reset();

  Release(place);
  CheckMemLeak(place);
}

TEST(StreamSafeCUDAAllocInterfaceTest, GetAllocatorInterfaceTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  auto &instance = allocation::AllocatorFacade::Instance();
  const std::shared_ptr<Allocator> &allocator = instance.GetAllocator(place);

  size_t alloc_size = 256;
  std::shared_ptr<Allocation> allocation_from_allocator =
      allocator->Allocate(alloc_size);
  EXPECT_GE(allocation_from_allocator->size(), alloc_size);
  void *address = allocation_from_allocator->ptr();
  allocation_from_allocator.reset();

  std::shared_ptr<Allocation> allocation_implicit_stream =
      AllocShared(place, alloc_size);
  EXPECT_GE(allocation_implicit_stream->size(), alloc_size);
  EXPECT_EQ(allocation_implicit_stream->ptr(), address);
  allocation_implicit_stream.reset();

  Release(place);
  CheckMemLeak(place);
}

TEST(StreamSafeCUDAAllocRetryTest, RetryTest) {
  platform::CUDAPlace place = platform::CUDAPlace();
  gpuStream_t stream1, stream2;
#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream1));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream2));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream1));
  PADDLE_ENFORCE_GPU_SUCCESS(hipStreamCreate(&stream2));
#endif
  size_t available_size = platform::GpuAvailableMemToAlloc();
  // alloc_size < available_size < 2 * alloc_size
  size_t alloc_size = available_size / 4 * 3;

  std::shared_ptr<Allocation> allocation1 =
      AllocShared(place, alloc_size, stream1);
  std::shared_ptr<Allocation> allocation2;

  std::thread th([&allocation2, &place, &stream2, alloc_size]() {
    std::this_thread::sleep_for(std::chrono::seconds(1));
    allocation2 = AllocShared(place, alloc_size, stream2);
  });
  allocation1.reset();  // free but not release
  th.join();
  EXPECT_GE(allocation2->size(), alloc_size);
  allocation2.reset();

#ifdef PADDLE_WITH_CUDA
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif

  Release(place, stream1);
  Release(place, stream2);
  CheckMemLeak(place);
}

}  // namespace memory
}  // namespace paddle
