#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <gtest/gtest.h>
#include <unordered_map>

#include "paddle/fluid/memory/detail/memory_block.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/memory/memory.h"

#include "paddle/fluid/platform/cpu_info.h"
#include "paddle/fluid/platform/gpu_info.h"
#include "paddle/fluid/platform/place.h"

// This unit test is an example comparing the performance between using pinned
// memory and not. In general, using pinned memory will be faster.
template <typename T>
__global__ void Kernel(T* output, int dim) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < dim) {
    output[tid] = output[tid] * output[tid] / 100;
  }
}

template <typename Place>
float test_pinned_memory() {
  Place cpu_place;
  paddle::platform::CUDAPlace cuda_place;

  const int data_size = 4096;
  const int iteration = 10;

  // create event start and end
  hipEvent_t start_e, stop_e, copying_e;
  float elapsedTime = 0;
  hipEventCreate(&start_e);
  hipEventCreate(&stop_e);
  hipEventCreate(&copying_e);

  // create computation stream, data copying stream
  hipStream_t computation_stream, copying_stream;
  hipStreamCreate(&computation_stream);
  hipStreamCreate(&copying_stream);

  // create record event, pinned memory, gpu memory
  std::vector<hipEvent_t> record_event(iteration);
  std::vector<float*> input_pinned_mem(iteration);
  std::vector<float*> gpu_mem(iteration);
  std::vector<float*> output_pinned_mem(iteration);

  // initial data
  for (int j = 0; j < iteration; ++j) {
    hipEventCreateWithFlags(&record_event[j], hipEventDisableTiming);
    hipEventCreate(&(record_event[j]));
    input_pinned_mem[j] = static_cast<float*>(
        paddle::memory::Alloc(cpu_place, data_size * sizeof(float)));
    output_pinned_mem[j] = static_cast<float*>(
        paddle::memory::Alloc(cpu_place, data_size * sizeof(float)));
    gpu_mem[j] = static_cast<float*>(
        paddle::memory::Alloc(cuda_place, data_size * sizeof(float)));

    for (int k = 0; k < data_size; ++k) {
      input_pinned_mem[j][k] = k;
    }
  }

  hipEventRecord(start_e, computation_stream);

  // computation
  for (int m = 0; m < 30; ++m) {
    for (int i = 0; i < iteration; ++i) {
      // cpu -> GPU on computation stream.
      // note: this operation is async for pinned memory.
      paddle::memory::Copy(cuda_place, gpu_mem[i], cpu_place,
                           input_pinned_mem[i], data_size * sizeof(float),
                           computation_stream);

      // call kernel on computation stream.
      Kernel<<<4, 1024, 0, computation_stream>>>(gpu_mem[i], data_size);

      // record event_computation on computation stream
      hipEventRecord(record_event[i], computation_stream);

      // wait event_computation on copy stream.
      // note: this operation is async.
      hipStreamWaitEvent(copying_stream, record_event[i], 0);

      // copy data GPU->CPU, on copy stream.
      // note: this operation is async for pinned memory.
      paddle::memory::Copy(cpu_place, output_pinned_mem[i], cuda_place,
                           gpu_mem[i], data_size * sizeof(float),
                           copying_stream);
    }
  }

  hipEventRecord(copying_e, copying_stream);
  hipStreamWaitEvent(computation_stream, copying_e, 0);

  hipEventRecord(stop_e, computation_stream);

  hipEventSynchronize(start_e);
  hipEventSynchronize(stop_e);
  hipEventElapsedTime(&elapsedTime, start_e, stop_e);

  // std::cout << cpu_place << " "
  //          << "time consume:" << elapsedTime / 30 << std::endl;

  for (int l = 0; l < iteration; ++l) {
    for (int k = 0; k < data_size; ++k) {
      float temp = input_pinned_mem[l][k];
      temp = temp * temp / 100;
      EXPECT_FLOAT_EQ(temp, output_pinned_mem[l][k]);
    }
  }

  // destroy resource
  hipEventDestroy(copying_e);
  hipEventDestroy(start_e);
  hipEventDestroy(stop_e);
  for (int j = 0; j < 10; ++j) {
    hipEventDestroy((record_event[j]));
    paddle::memory::Free(cpu_place, input_pinned_mem[j]);
    paddle::memory::Free(cpu_place, output_pinned_mem[j]);
    paddle::memory::Free(cuda_place, gpu_mem[j]);
  }
  return elapsedTime / 30;
}

TEST(CPUANDCUDAPinned, CPUAllocatorAndCUDAPinnedAllocator) {
  // Generally speaking, operation on pinned_memory is faster than that on
  // unpinned-memory, but if this unit test fails frequently, please close this
  // test for the time being.
  float time1 = test_pinned_memory<paddle::platform::CPUPlace>();
  float time2 = test_pinned_memory<paddle::platform::CUDAPinnedPlace>();
  EXPECT_GT(time1, time2);
}
