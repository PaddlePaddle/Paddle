#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/conv_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/scatter.cu.h"
#include "paddle/phi/kernels/funcs/sparse/scatter.cu.h"
#include "paddle/phi/kernels/sparse/gpu/conv.cu.h"

#include "glog/logging.h"

namespace phi {
namespace sparse {

#if 0
enum Type { kRule, kNnz, kIdx, kNormalInt, kNormalFloat };
template <typename IntT>
__global__ void print(const IntT* p, int len, Type t = kRule) {
  if (t == kNnz) {
    for (int i = 0; i < len; i++) {
      printf("%f,", *(p + i));
      if ((i + 1) % 2 == 0) printf("\n");
    }
    printf("\n");
  }

  if (t == kIdx) {
    for (int i = 0; i < len; i++) {
      printf("%d,%d,%d,%d\n",
             *(p + i),
             *(p + len + i),
             *(p + 2 * len + i),
             *(p + 3 * len + i));
    }
  }

  if (t == kRule) {
    for (int i = 0; i < len; i++) {
      printf("%d,%d\n", *(p + i), *(p + len + i));
    }
  }

  if (t == kNormalInt) {
    for (int i = 0; i < len; i++) {
      printf("%d,", *(p + i));
      if ((i + 1) % 8 == 0) printf("\n");
    }
    printf("\n");
  }

  if (t == kNormalFloat) {
    for (int i = 0; i < len; i++) {
      printf("%f,", *(p + i));
      if ((i + 1) % 10 == 0) printf("\n");
    }
  }
}
#endif

template <typename T, typename IntT>
void Conv3dCooGPUKernel(const GPUContext& dev_ctx,
                        const SparseCooTensor& x,
                        const DenseTensor& kernel,
                        const std::vector<int>& paddings,
                        const std::vector<int>& dilations,
                        const std::vector<int>& strides,
                        const int groups,
                        const bool subm,
                        const std::string& key,
                        SparseCooTensor* out,
                        DenseTensor* rulebook,
                        DenseTensor* counter) {
  // update padding and dilation
  // Currently, only support x.layout is NDHWC, groups = 1
  // if x.layout != NDHWC then transpose(x), transpose(weight)
  const auto& x_dims = x.dims();
  const auto& kernel_dims = kernel.dims();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  DDim out_dims = {1, 1, 1, 1, 1};
  std::vector<int> kernel_sizes(kernel_dims.size());
  for (int i = 0; i < kernel_dims.size(); i++) {
    kernel_sizes[i] = kernel_dims[i];
  }

  std::vector<int> subm_paddings(paddings), subm_strides(strides);
  if (subm) {
    // the out shape of subm_conv is same as input shape
    // reset the padding=kernel_size/2 and strides=1
    phi::funcs::sparse::ResetSubmKernelSizeAndStrides(
        kernel.dims(), &subm_paddings, &subm_strides);
  }

  phi::funcs::sparse::GetOutShape(
      x_dims, kernel_sizes, subm_paddings, dilations, subm_strides, &out_dims);
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];
  DenseTensor h_counter, h_offsets;
  h_counter.Resize({kernel_size});
  h_offsets.Resize({kernel_size + 1});
  int* h_counter_ptr = dev_ctx.template HostAlloc<int>(&h_counter);
  int* h_offsets_ptr = dev_ctx.template HostAlloc<int>(&h_offsets);

  // Second algorithm:
  // https://pdfs.semanticscholar.org/5125/a16039cabc6320c908a4764f32596e018ad3.pdf
  // 1. product rulebook
  DenseTensor counter_per_kernel = phi::Empty<int>(dev_ctx, {kernel_size});
  DenseTensor offsets_per_kernel = phi::Empty<int>(dev_ctx, {kernel_size});
  DenseTensor out_index = phi::Empty<int>(dev_ctx, {1});
  DenseTensor unique_value = phi::Empty<int>(dev_ctx, {1});

  VLOG(6) << "call SubmConv3D or Conv3D " << subm << " and the key is " << key;
  int rulebook_len = 0;
  const IntT* rulebook_ptr = nullptr;
  bool need_product_rulebook = true;
  if (subm && !key.empty()) {
    rulebook_ptr = phi::funcs::sparse::PrepareSubm<T, IntT, GPUContext>(
        dev_ctx,
        x,
        key,
        out_dims,
        out,
        h_counter.data<int>(),
        h_offsets.data<int>(),
        &rulebook_len,
        &need_product_rulebook);
  }

  if (need_product_rulebook) {
    DenseTensor tmp_rulebook;
    rulebook_len = ProductRuleBook<T, GPUContext, IntT>(dev_ctx,
                                                        x,
                                                        kernel_sizes,
                                                        subm_paddings,
                                                        dilations,
                                                        subm_strides,
                                                        out_dims,
                                                        subm,
                                                        &tmp_rulebook,
                                                        &counter_per_kernel,
                                                        &offsets_per_kernel,
                                                        &out_index,
                                                        &unique_value,
                                                        out,
                                                        h_counter_ptr,
                                                        h_offsets_ptr);
    rulebook_ptr = tmp_rulebook.data<IntT>();

#if 0
    std::cout << "counter,offset:" << std::endl;
    for (int i = 0; i < kernel_size; i++) {
      printf("%d,%d\n", h_counter_ptr[i], h_offsets_ptr[i]);
  }
  printf("\n");

  std::cout<<"rulebook len: "<<rulebook_len<<std::endl;
  std::cout<<"tmp size: "<<tmp_rulebook.dims().size()<<std::endl;
  std::cout<<"tmp: "<<tmp_rulebook.dims().at(0)<<","<<tmp_rulebook.dims().at(1)<<std::endl;

  std::cout<<"tmp value:"<<std::endl;
  print<<<1,1>>>(tmp_rulebook.data<IntT>(),(int)tmp_rulebook.dims().at(1),kRule);
  hipDeviceSynchronize();
#endif

    phi::funcs::sparse::SaveToTable(
        dev_ctx, x, key, tmp_rulebook, h_counter, out, rulebook, counter);
  }

#if 0
  std::cout<<"rule size: "<<rulebook->dims().size()<<std::endl;
  std::cout<<"rule : "<<rulebook->dims().at(0)<<","<<rulebook->dims().at(1)<<std::endl;

  std::cout<<"rulebook value:"<<std::endl;
  print<<<1,1>>>(rulebook->data<IntT>(),(int)rulebook->dims().at(1),kRule);
  hipDeviceSynchronize();

  std::cout<<"nnz value:"<<x.non_zero_elements().numel()<<std::endl;
  print<<<1,1>>>(x.non_zero_elements().data<T>(),x.non_zero_elements().numel(),kNnz);
  hipDeviceSynchronize();
  std::cout<<"nnz idx:"<<x.non_zero_indices().numel()/4<<std::endl;
  print<<<1,1>>>(x.non_zero_indices().data<IntT>(),x.non_zero_indices().numel()/4,kIdx);
  hipDeviceSynchronize();
#endif

  auto* out_values = out->mutable_non_zero_elements();
  T* out_values_ptr = out_values->data<T>();
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  set_zero(dev_ctx, out_values, static_cast<T>(0.0f));

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (h_counter_ptr[i] <= 0) {
      continue;
    }

    const int M = h_counter_ptr[i];
    const int K = in_channels;
    const int N = out_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * K * N;
    const IntT* gather_indices = rulebook_ptr+h_offsets_ptr[i];
    const IntT* scatter_indices = rulebook_ptr+rulebook_len+h_offsets_ptr[i];

    gather_gemm_scatter<T, T, T, T, T>(x.non_zero_elements().data<T>(),
                                       tmp_kernel_ptr,
                                       out_values_ptr,
                                       out_values_ptr,
                                       M,
                                       N,
                                       K,
                                       gather_indices,
                                       scatter_indices,
                                       h_counter_ptr[i],
                                       static_cast<T>(1),
                                       static_cast<T>(1));
  }

#if 0
  // 2. gather
  phi::DenseTensor in_features =
      phi::Empty<T>(dev_ctx, {rulebook_len, in_channels});
  phi::DenseTensor out_features =
      phi::Empty<T>(dev_ctx, {rulebook_len, out_channels});
  T* in_features_ptr = in_features.data<T>();
  T* out_features_ptr = out_features.data<T>();
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  set_zero(dev_ctx, &out_features, static_cast<T>(0.0f));

  Gather<T, IntT>(dev_ctx,
                  x.non_zero_elements().data<T>(),
                  rulebook_ptr,
                  rulebook_len,
                  in_channels,
                  in_features_ptr);

  // 3. call gemm for every werght
  auto blas = phi::funcs::GetBlas<GPUContext, T>(dev_ctx);
  auto* out_values = out->mutable_non_zero_elements();
  T* out_values_ptr = out_values->data<T>();
  set_zero(dev_ctx, out_values, static_cast<T>(0.0f));

  if (subm) {
    auto config =
        phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, rulebook_len, 1);
    unique_value.ResizeAndAllocate(
        {static_cast<int>(out->nnz() * kernel_size)});
    out_index.ResizeAndAllocate({static_cast<int>(rulebook_len)});
    int* out_index_ptr = out_index.data<int>();
    int* unique_value_ptr = unique_value.data<int>();
    phi::backends::gpu::GpuMemsetAsync(
        out_index_ptr, 0, sizeof(int) * rulebook_len, dev_ctx.stream());
    GroupIndexs<<<config.block_per_grid,
                  config.thread_per_block,
                  0,
                  dev_ctx.stream()>>>(rulebook_len,
                                      kernel_size,
                                      rulebook_ptr + rulebook_len,
                                      out_index_ptr,
                                      unique_value_ptr);
  }

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (h_counter_ptr[i] <= 0) {
      continue;
    }

    // call gemm: (n, in_channels) * (in_channels, out_channels)
    const int M = h_counter_ptr[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + h_offsets_ptr[i] * in_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * K * N;
    T* tmp_out_ptr = out_features_ptr + h_offsets_ptr[i] * out_channels;

    blas.GEMM(CblasNoTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_out_ptr);
  }

#if 0
    std::cout<<"out_indices:" << out->non_zero_indices().numel() / 4 << std::endl;
    print<<<1, 1>>>(out->non_zero_indices().data<IntT>(), out->non_zero_indices().numel() / 4, kIdx);
    hipDeviceSynchronize();

    std::cout << "out_index:" << out_index.numel() << std::endl;
    print<<<1, 1>>>(out_index.data<int>(), out_index.numel(), kNormalInt);
    hipDeviceSynchronize();

    std::cout << "unique_value:" << unique_value.numel() << std::endl;
    print<<<1, 1>>>(
        unique_value.data<int>(), unique_value.numel(), kNormalInt);
    hipDeviceSynchronize();

    std::cout << "out_features:" << out_features.numel() << std::endl;
    print<<<1, 1>>>(out_features.data<T>(), out_features.numel(), kNnz);
    hipDeviceSynchronize();
#endif
  // 4. scatter
  phi::funcs::sparse::ScatterV2<T>(dev_ctx,
                                   out_features_ptr,
                                   out_index.data<int>(),
                                   unique_value.data<int>(),
                                   out->nnz(),
                                   kernel_size,
                                   out_channels,
                                   1,
                                   out_values_ptr);
#if 0
  hipDeviceSynchronize();
  std::cout << "out:" << out_values->numel() << std::endl;
  print<<<1, 1>>>(out_values->data<T>(), out_values->numel(), kNnz);
  hipDeviceSynchronize();
#endif
#endif
}

/**
 * x: the input SparseCooTensor, shape is (N, D, H, W, C)
 * kernel: the weight data, shape is (D, H, W, C, OC)
 * out: the output SparseCooTensor, shape is (N, D, H, W, OC)
 * rulebook: return rulebook if key is not vailed else return nullptr
 * counter: return counter if key is not vailed else return nullptr
 **/
template <typename T, typename Context>
void Conv3dCooKernel(const Context& dev_ctx,
                     const SparseCooTensor& x,
                     const DenseTensor& kernel,
                     const std::vector<int>& paddings,
                     const std::vector<int>& dilations,
                     const std::vector<int>& strides,
                     const int groups,
                     const bool subm,
                     const std::string& key,
                     SparseCooTensor* out,
                     DenseTensor* rulebook,
                     DenseTensor* counter) {
#if 0
  PD_VISIT_INTEGRAL_TYPES(
      x.non_zero_indices().dtype(), "Conv3dCooGPUKernel", ([&] {
#endif
        Conv3dCooGPUKernel<T, int32_t>(dev_ctx,
                                      x,
                                      kernel,
                                      paddings,
                                      dilations,
                                      strides,
                                      groups,
                                      subm,
                                      key,
                                      out,
                                      rulebook,
                                      counter);
      #if 0
      }));
      #endif
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(conv3d_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dCooKernel,
                   float) {
#if 0
                   double,
                   phi::dtype::float16) {
#endif
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
