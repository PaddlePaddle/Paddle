/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/conv_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/scatter.cu.h"
#include "paddle/phi/kernels/funcs/sparse/scatter.cu.h"
#include "paddle/phi/kernels/sparse/gpu/convolution.cu.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT>
void Conv3dCooGPUKernel(const GPUContext& dev_ctx,
                        const SparseCooTensor& x,
                        const DenseTensor& kernel,
                        const std::vector<int>& paddings,
                        const std::vector<int>& dilations,
                        const std::vector<int>& strides,
                        const int groups,
                        const bool subm,
                        SparseCooTensor* out,
                        DenseTensor* rulebook) {
  // update padding and dilation
  // Currently, only support x.layout is NDHWC, groups = 1
  // if x.layout != NDHWC then transpose(x), transpose(weight)
  const auto& x_dims = x.dims();
  const auto& kernel_dims = kernel.dims();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  DDim out_dims = {1, 1, 1, 1, 1};
  std::vector<int> kernel_sizes(kernel_dims.size());
  for (int i = 0; i < kernel_dims.size(); i++) {
    kernel_sizes[i] = kernel_dims[i];
  }

  std::vector<int> subm_paddings(paddings), subm_strides(strides);
  if (subm) {
    // the out shape of subm_conv is same as input shape
    // reset the padding=kernel_size/2 and strides=1
    phi::funcs::sparse::ResetSubmKernelSizeAndStrides(
        kernel.dims(), &subm_paddings, &subm_strides);
  }

  phi::funcs::sparse::GetOutShape(
      x_dims, kernel_sizes, subm_paddings, dilations, subm_strides, &out_dims);
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];
  std::vector<int> offsets(kernel_size + 1), h_counter(kernel_size);

  // Second algorithm:
  // https://pdfs.semanticscholar.org/5125/a16039cabc6320c908a4764f32596e018ad3.pdf
  // 1. product rulebook
  DenseTensorMeta counter_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensorMeta offsets_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensor counter_per_kernel = phi::Empty(dev_ctx, std::move(counter_meta));
  DenseTensor offsets_per_kernel = phi::Empty(dev_ctx, std::move(offsets_meta));
  DenseTensorMeta index_meta(DataType::INT32, {1}, DataLayout::NCHW);
  DenseTensor out_index = phi::Empty(dev_ctx, std::move(index_meta));
  DenseTensor unique_value = phi::Empty(dev_ctx, std::move(index_meta));

  int n = ProductRuleBook<T, GPUContext, IntT>(dev_ctx,
                                               x,
                                               kernel_sizes,
                                               subm_paddings,
                                               dilations,
                                               subm_strides,
                                               out_dims,
                                               subm,
                                               rulebook,
                                               &counter_per_kernel,
                                               &offsets_per_kernel,
                                               &out_index,
                                               &unique_value,
                                               out,
                                               &h_counter,
                                               &offsets);

  const int* counter_ptr = counter_per_kernel.data<int>();
  const int* offsets_ptr = counter_per_kernel.data<int>();
  const IntT* rulebook_ptr = rulebook->data<IntT>();

  // 2. gather
  DenseTensorMeta in_features_meta(
      x.dtype(), {n, in_channels}, DataLayout::NCHW);
  DenseTensorMeta out_features_meta(
      x.dtype(), {n, out_channels}, DataLayout::NCHW);
  phi::DenseTensor in_features =
      phi::Empty(dev_ctx, std::move(in_features_meta));
  phi::DenseTensor out_features =
      phi::Empty(dev_ctx, std::move(out_features_meta));
  T* in_features_ptr = in_features.data<T>();
  T* out_features_ptr = out_features.data<T>();
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  set_zero(dev_ctx, &out_features, static_cast<T>(0.0f));

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n * in_channels, 1);
  GatherKernel<T, IntT><<<config.block_per_grid.x,
                          config.thread_per_block.x,
                          0,
                          dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                                              rulebook_ptr + n,
                                              in_features_ptr,
                                              n,
                                              in_channels);

  // 3. call gemm for every werght
  auto blas = phi::funcs::GetBlas<GPUContext, T>(dev_ctx);
  auto* out_values = out->mutable_non_zero_elements();
  T* out_values_ptr = out_values->data<T>();

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (h_counter[i] <= 0) {
      continue;
    }

    // call gemm: (n, in_channels) * (in_channels, out_channels)
    const int M = h_counter[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + offsets[i] * in_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * K * N;
    T* tmp_out_ptr = out_features_ptr + offsets[i] * out_channels;

    blas.GEMM(CblasNoTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_out_ptr);
  }

  // 4. scatter
  if (subm) {
    set_zero(dev_ctx, out_values, static_cast<T>(0.0f));
    config =
        phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n * out_channels, 1);
    phi::funcs::ScatterCUDAKernel<T, IntT>
        <<<config.block_per_grid,
           config.thread_per_block,
           0,
           dev_ctx.stream()>>>(out_features_ptr,
                               rulebook_ptr + 2 * n,
                               out_values_ptr,
                               n,
                               out_channels,
                               false);
  } else {
    config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, out->nnz() * out_channels, 1);
    phi::funcs::sparse::ScatterKernel<T>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(out_features_ptr,
                               unique_value.data<int>(),
                               out_index.data<int>(),
                               out->nnz(),
                               n,
                               out_channels,
                               out_values_ptr);
  }
}
/**
 * x: (N, D, H, W, C)
 * kernel: (D, H, W, C, OC)
 * out: (N, D, H, W, OC)
 **/
template <typename T, typename Context>
void Conv3dCooKernel(const Context& dev_ctx,
                     const SparseCooTensor& x,
                     const DenseTensor& kernel,
                     const std::vector<int>& paddings,
                     const std::vector<int>& dilations,
                     const std::vector<int>& strides,
                     const int groups,
                     const bool subm,
                     SparseCooTensor* out,
                     DenseTensor* rulebook) {
  PD_VISIT_INTEGRAL_TYPES(
      x.non_zero_indices().dtype(), "Conv3dCooGPUKernel", ([&] {
        Conv3dCooGPUKernel<T, data_t>(dev_ctx,
                                      x,
                                      kernel,
                                      paddings,
                                      dilations,
                                      strides,
                                      groups,
                                      subm,
                                      out,
                                      rulebook);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(conv3d_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dCooKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
