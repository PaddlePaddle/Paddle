/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/addmm_kernel.h"

#include <vector>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/sparse/sparse_blas.h"

namespace phi {
namespace sparse {

template <typename T, typename Context, typename TensorType>
void AddmmKernelImpl(const Context& dev_ctx,
                     const DenseTensor& input,
                     const TensorType& x,
                     const DenseTensor& y,
                     float alpha,
                     float beta,
                     DenseTensor* out) {
#if CUDA_VERSION >= 11000
  std::vector<int64_t> input_dim = phi::vectorize(input.dims());
  std::vector<int64_t> x_dim = phi::vectorize(x.dims());
  std::vector<int64_t> y_dim = phi::vectorize(y.dims());
  auto rank = input_dim.size();

  PADDLE_ENFORCE_GE(
      rank,
      2,
      phi::errors::InvalidArgument(
          "the dims size of input must be greater than or eaqual to 2."));

  PADDLE_ENFORCE_EQ(
      x_dim.size(),
      rank,
      phi::errors::PreconditionNotMet(
          "The dims size of Input(input) and Input(x) must be eaqual."));

  PADDLE_ENFORCE_GE(
      y_dim.size(),
      rank,
      phi::errors::InvalidArgument(
          "the dims size of Input(input) and Input(y) must be eaqual."));

  for (size_t i = 0; i < rank - 2; ++i) {
    PADDLE_ENFORCE_EQ(input_dim[i],
                      x_dim[i],
                      phi::errors::InvalidArgument(
                          "input.dim[%d] and x.dim[%d] must be eaqul.", i, i));
    PADDLE_ENFORCE_EQ(input_dim[i],
                      y_dim[i],
                      phi::errors::InvalidArgument(
                          "input.dim[%d] and y.dim[%d] must be eaqul.", i, i));
  }

  PADDLE_ENFORCE_GE(
      input_dim[rank - 2],
      x_dim[rank - 2],
      phi::errors::PreconditionNotMet(
          "The shape of Input(input) and Input(x) is not suitable for matmul "
          "opetation, input_dim[-2] must be eaqual to x_dim[-2]."));

  PADDLE_ENFORCE_GE(
      input_dim[rank - 1],
      y_dim[rank - 1],
      phi::errors::PreconditionNotMet(
          "The shape of Input(input) and Input(y) is not suitable for matmul "
          "opetation, input_dim[-1] must be eaqual to y_dim[-1]."));

  PADDLE_ENFORCE_GE(
      x_dim[rank - 1],
      y_dim[rank - 2],
      phi::errors::PreconditionNotMet(
          "The shape of Input(x) and Input(y) is not suitable for matmul "
          "opetation, x_dim[-1] must be eaqual to y_dim[-2]."));

  phi::Copy(dev_ctx, input, dev_ctx.GetPlace(), false, out);

  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);
  sparse_blas.SPMM(
      false, false, static_cast<T>(alpha), x, y, static_cast<T>(beta), out);
#else
  PADDLE_THROW(
      phi::errors::Unimplemented("forward of 'sparse.addmm' use hipsparseSpMM, "
                                 "which is supported from CUDA 11.0"));
#endif
}

template <typename T, typename Context>
void AddmmCooDenseKernel(const Context& dev_ctx,
                         const DenseTensor& input,
                         const SparseCooTensor& x,
                         const DenseTensor& y,
                         float alpha,
                         float beta,
                         DenseTensor* out) {
  AddmmKernelImpl<T>(dev_ctx, input, x, y, alpha, beta, out);
}

template <typename T, typename Context>
void AddmmCsrDenseKernel(const Context& dev_ctx,
                         const DenseTensor& input,
                         const SparseCsrTensor& x,
                         const DenseTensor& y,
                         float alpha,
                         float beta,
                         DenseTensor* out) {
  AddmmKernelImpl<T>(dev_ctx, input, x, y, alpha, beta, out);
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(addmm_coo_dense,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::AddmmCooDenseKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}

PD_REGISTER_KERNEL(addmm_csr_dense,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::AddmmCsrDenseKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}
