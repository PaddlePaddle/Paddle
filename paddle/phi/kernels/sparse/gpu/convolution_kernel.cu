#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "paddle/phi/api/lib/utils/allocator.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/primitive/compute_primitives.h"
#include "paddle/phi/kernels/sparse/convolution_kernel.h"

namespace phi {
namespace sparse {

// TODO(zhangkaihuo) replace this kernel with KP::InitWithDataIndex
__global__ void InitByIndexKernel(const int n, int* out1, int* out2) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += gridDim.x * blockDim.x) {
    out1[i] = i;
    out2[i] = i;
  }
}

__global__ void UpdateIndexKernel(const int* unique_keys,
                                  const int* unique_values,
                                  const int* out_indexs,
                                  const int non_zero_num,
                                  const int rulebook_len,
                                  const Dims4D out_dims,
                                  int* out_indices,
                                  int* rulebook_out_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    const int index = unique_keys[i];
    int batch, x, y, z;
    IndexToPoint<Dims4D>(index, out_dims, &batch, &x, &y, &z);
    // get out indices
    out_indices[i] = batch;
    out_indices[i + non_zero_num] = z;
    out_indices[i + non_zero_num * 2] = y;
    out_indices[i + non_zero_num * 3] = x;

    // update rulebook
    int start = unique_values[i];
    int end = i == non_zero_num - 1 ? rulebook_len : unique_values[i + 1];
    // max(end-start) = kernel_size
    for (int j = start; j < end; j++) {
      rulebook_out_indexs[out_indexs[j]] = i;
    }
  }
}

__global__ void ProductRuleBookKernel(const int* x_indices,
                                      const Dims4D x_dims,
                                      const Dims4D kernel_dims,
                                      const Dims4D out_dims,
                                      const int64_t non_zero_num,
                                      const Dims4D paddings,
                                      const Dims4D dilations,
                                      const Dims4D strides,
                                      int* rulebook,
                                      int* counter) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  extern __shared__ int counter_buf[];  // kernel_size
  const int kernel_size = kernel_dims[3] * kernel_dims[2] * kernel_dims[1];
  const int offset = kernel_size * non_zero_num;
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    counter_buf[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int kernel_index = 0;
    for (int kz = 0; kz < kernel_dims[1]; kz++) {
      for (int ky = 0; ky < kernel_dims[2]; ky++) {
        for (int kx = 0; kx < kernel_dims[3]; kx++) {
          int batch = x_indices[i];
          int in_z = x_indices[i + non_zero_num];
          int in_y = x_indices[i + 2 * non_zero_num];
          int in_x = x_indices[i + 3 * non_zero_num];
          int in_i = -1, out_index = -1;
          if (Check(x_dims,
                    kernel_dims,
                    paddings,
                    dilations,
                    strides,
                    in_x,
                    in_y,
                    in_z,
                    kx,
                    ky,
                    kz)) {
            int out_z = (in_z + paddings[1] - kz * dilations[1]) / strides[1];
            int out_y = (in_y + paddings[2] - ky * dilations[2]) / strides[2];
            int out_x = (in_x + paddings[3] - kx * dilations[3]) / strides[3];
            in_i = i;
            out_index =
                PointToIndex<Dims4D>(batch, out_x, out_y, out_z, out_dims);
            atomicAdd(&counter_buf[kernel_index], 1);
          }
          rulebook[kernel_index * non_zero_num + i] = in_i;
          rulebook[kernel_index * non_zero_num + offset + i] = out_index;
          ++kernel_index;
        }
      }
    }
  }
  __syncthreads();
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    atomicAdd(&counter[i], counter_buf[i]);
  }
}

// TODO(zhangkaihuo): After the GatherCUDAKernel is migrated to phi, replace
// this kernel with phi::GatherCUDAKernel;
template <typename T, typename IndexT = int>
__global__ void GatherKernel(const T* params,
                             const IndexT* indices,
                             T* output,
                             size_t index_size,
                             size_t slice_size) {
  CUDA_KERNEL_LOOP_TYPE(i, index_size * slice_size, int64_t) {
    int64_t indices_i = i / slice_size;
    int64_t slice_i = i - indices_i * slice_size;  // offset inside the slice
    IndexT gather_i = indices[indices_i];
    int64_t params_i = gather_i * slice_size + slice_i;
    *(output + i) = *(params + params_i);
  }
}

template <typename T>
__global__ void ScatterKernel(const T* input,
                              const int* unique_value,
                              const int* out_index,
                              const int non_zero_num,
                              const int rulebook_len,
                              const int channels,
                              T* out) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num * channels; i += gridDim.x * blockDim.x) {
    int indices_i = i / channels;
    int channels_i = i - indices_i * channels;

    int start = unique_value[indices_i];
    int end = indices_i == non_zero_num - 1 ? rulebook_len
                                            : unique_value[indices_i + 1];
    // max(end-start) = kernel_size
    T sum = static_cast<T>(0);
    for (int j = start; j < end; j++) {
      const int out_feature_i = out_index[j];
      sum += input[out_feature_i * channels + channels_i];
    }
    out[indices_i * channels + channels_i] = sum;
  }
}

// the basic algorithm can refer to convolution_kernel.cc or
// the second paper
// example:
// 1. the rulebook:
//  the kernel_index:               0, 0, 0, 1, 1, 1, 2, 2, ....
//  the out_index(key):                  20, 30, 33, 30, 33, 20, 25
// 2. mark the index of out_index(value):   0, 1, 2, 3, 4, 5, 6, ....
// 3. sorted the (key, value)
// 4. unique the (key, value):
//  unique_key:     20, 25, 30, 33
//  unique_values:  0, 2, 3, 5
//  the index of unique_values is: 0, 1, 2, 3
// 5. update the out_index by unique_key, uniqe_value and the index of
// unique_value:
//  the new out_index: 0, 2, 3, 2, 3, 0, 1
template <typename T, typename Context>
int ProductRuleBook(const Context& dev_ctx,
                    const SparseCooTensor& x,
                    const DenseTensor& kernel,
                    const std::vector<int>& paddings,
                    const std::vector<int>& dilations,
                    const std::vector<int>& strides,
                    const DDim& out_dims,
                    DenseTensor* rulebook,
                    DenseTensor* counter_per_kernel,
                    DenseTensor* offsets_per_kernel,
                    DenseTensor* out_index,
                    DenseTensor* unique_key,
                    DenseTensor* unique_value,
                    SparseCooTensor* out,
                    std::vector<int>* h_counter,
                    std::vector<int>* h_offsets) {
  // const auto place = dev_ctx.GetPlace();
  const auto& kernel_dims = kernel.dims();
  const int64_t non_zero_num = x.nnz();
  const auto& non_zero_indices = x.non_zero_indices();
  const int* indices_ptr = non_zero_indices.data<int>();
  // int* counter_ptr = counter_per_kernel->mutable_data<int>(place);
  dev_ctx.Alloc(counter_per_kernel,
                counter_per_kernel->dtype(),
                sizeof(int) * counter_per_kernel->numel());
  int* counter_ptr = counter_per_kernel->data<int>();
  // int* offsets_ptr = offsets_per_kernel->mutable_data<int>(place);
  dev_ctx.Alloc(offsets_per_kernel,
                offsets_per_kernel->dtype(),
                sizeof(int) * offsets_per_kernel->numel());
  int* offsets_ptr = offsets_per_kernel->data<int>();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  rulebook->ResizeAndAllocate({2 * kernel_size * non_zero_num});
  // int* rulebook_ptr = rulebook->mutable_data<int>(place);
  dev_ctx.Alloc(rulebook, rulebook->dtype(), sizeof(int) * rulebook->numel());
  int* rulebook_ptr = rulebook->data<int>();

  const auto x_dims = x.dims();
  Dims4D d_x_dims(x_dims[0], x_dims[3], x_dims[2], x_dims[1]);
  Dims4D d_kernel_dims(1, kernel_dims[2], kernel_dims[1], kernel_dims[0]);
  Dims4D d_out_dims(out_dims[0], out_dims[3], out_dims[2], out_dims[1]);
  Dims4D d_paddings(1, paddings[2], paddings[1], paddings[0]);
  Dims4D d_strides(1, strides[2], strides[1], strides[0]);
  Dims4D d_dilations(1, dilations[2], dilations[1], dilations[0]);

  // 1. product rule book
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemsetAsync(
      counter_ptr, 0, sizeof(int) * kernel_size, dev_ctx.stream()));
  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, non_zero_num, 1);

  ProductRuleBookKernel<<<config.block_per_grid.x,
                          config.thread_per_block.x,
                          kernel_size * sizeof(int),
                          dev_ctx.stream()>>>(indices_ptr,
                                              d_x_dims,
                                              d_kernel_dims,
                                              d_out_dims,
                                              non_zero_num,
                                              d_paddings,
                                              d_dilations,
                                              d_strides,
                                              rulebook_ptr,
                                              counter_ptr);

  // 2. remove -1
  int* last = thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
                             rulebook_ptr,
                             rulebook_ptr + 2 * kernel_size * non_zero_num,
                             -1);
  thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
                         counter_ptr,
                         counter_ptr + kernel_size,
                         offsets_ptr);

  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&(*h_counter)[0],
                                             counter_ptr,
                                             kernel_size * sizeof(int),
                                             hipMemcpyDeviceToHost,
                                             dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&(*h_offsets)[0],
                                             offsets_ptr,
                                             kernel_size * sizeof(int),
                                             hipMemcpyDeviceToHost,
                                             dev_ctx.stream()));
  dev_ctx.Wait();
  int rulebook_len =
      (*h_counter)[kernel_size - 1] + (*h_offsets)[kernel_size - 1];

  // 3. sorted or merge the out index
  out_index->ResizeAndAllocate({rulebook_len});
  unique_value->ResizeAndAllocate({rulebook_len});
  unique_key->ResizeAndAllocate({rulebook_len});
  // int* out_index_ptr = out_index->mutable_data<int>(place);
  dev_ctx.Alloc(
      out_index, out_index->dtype(), sizeof(int) * out_index->numel());
  int* out_index_ptr = out_index->data<int>();
  // int* unique_value_ptr = unique_value->mutable_data<int>(place);
  dev_ctx.Alloc(
      unique_value, unique_value->dtype(), sizeof(int) * unique_value->numel());
  int* unique_value_ptr = unique_value->data<int>();
  // int* unique_key_ptr = unique_key->mutable_data<int>(place);
  dev_ctx.Alloc(
      unique_key, unique_key->dtype(), sizeof(int) * unique_key->numel());
  int* unique_key_ptr = unique_key->data<int>();

  config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, rulebook_len, 1);
  InitByIndexKernel<<<config.block_per_grid.x,
                      config.thread_per_block.x,
                      0,
                      dev_ctx.stream()>>>(
      rulebook_len, out_index_ptr, unique_value_ptr);

  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(unique_key_ptr,
                                             rulebook_ptr + rulebook_len,
                                             sizeof(int) * rulebook_len,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));

  // compared with thrust::sort_by_key, thrust::merge_by_key may achieved higher
  // performance, but thrust::merge_by_key limited by data size
  thrust::sort_by_key(thrust::cuda::par.on(dev_ctx.stream()),
                      unique_key_ptr,
                      unique_key_ptr + rulebook_len,
                      out_index_ptr);

  // 4. unique => tmp2_out_index
  thrust::pair<int*, int*> new_end =
      thrust::unique_by_key(thrust::cuda::par.on(dev_ctx.stream()),
                            unique_key_ptr,
                            unique_key_ptr + rulebook_len,
                            unique_value_ptr);
  dev_ctx.Wait();
  const int out_non_zero_num = thrust::distance(unique_key_ptr, new_end.first);

  // 5. update out_indices and rulebook by unique_value_ptr
  const int64_t sparse_dim = 4;
  DenseTensorMeta indices_meta(
      DataType::INT32, {sparse_dim, out_non_zero_num}, DataLayout::NCHW);
  DenseTensorMeta values_meta(
      x.dtype(), {out_non_zero_num, kernel_dims[4]}, x.layout());
  phi::DenseTensor out_indices = phi::Empty(dev_ctx, std::move(indices_meta));
  phi::DenseTensor out_values = phi::Empty(dev_ctx, std::move(values_meta));
  // int* out_indices_ptr = out_indices.mutable_data<int>(dev_ctx.GetPlace());
  dev_ctx.Alloc(
      &out_indices, out_indices.dtype(), sizeof(int) * out_indices.numel());
  int* out_indices_ptr = out_indices.data<int>();
  config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, out_non_zero_num, 1);
  UpdateIndexKernel<<<config.block_per_grid.x,
                      config.thread_per_block.x,
                      0,
                      dev_ctx.stream()>>>(unique_key_ptr,
                                          unique_value_ptr,
                                          out_index_ptr,
                                          out_non_zero_num,
                                          rulebook_len,
                                          d_out_dims,
                                          out_indices_ptr,
                                          rulebook_ptr + rulebook_len);
  out->SetMember(out_indices, out_values, out_dims, true);
  return rulebook_len;
}

/**
 * x: (N, D, H, W, C)
 * kernel: (D, H, W, C, OC)
 * out: (N, D, H, W, OC)
**/
template <typename T, typename Context>
void Conv3dKernel(const Context& dev_ctx,
                  const SparseCooTensor& x,
                  const DenseTensor& kernel,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const int groups,
                  SparseCooTensor* out,
                  DenseTensor* rulebook) {
  // update padding and dilation
  // Currently, only support x.layout is NDHWC, groups = 1
  // if x.layout != NDHWC then transpose(x), transpose(weight)

  const auto& place = dev_ctx.GetPlace();
  const auto& x_dims = x.dims();
  const auto& kernel_dims = kernel.dims();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  DDim out_dims = {1, 1, 1, 1, 1};
  GetOutShape(x_dims, kernel_dims, paddings, dilations, strides, &out_dims);
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];
  std::vector<int> offsets(kernel_size + 1), h_counter(kernel_size);

  // Second algorithm:
  // https://pdfs.semanticscholar.org/5125/a16039cabc6320c908a4764f32596e018ad3.pdf
  // 1. product rulebook
  DenseTensorMeta counter_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensorMeta offsets_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  // DenseTensor rulebook = phi::Empty<int, Context>(dev_ctx);
  DenseTensor counter_per_kernel = phi::Empty(dev_ctx, std::move(counter_meta));
  DenseTensor offsets_per_kernel = phi::Empty(dev_ctx, std::move(offsets_meta));
  DenseTensor out_index = phi::Empty<int, Context>(dev_ctx);
  DenseTensor unique_key = phi::Empty<int, Context>(dev_ctx);
  DenseTensor unique_value = phi::Empty<int, Context>(dev_ctx);

  int n = ProductRuleBook<T, Context>(dev_ctx,
                                      x,
                                      kernel,
                                      paddings,
                                      dilations,
                                      strides,
                                      out_dims,
                                      rulebook,
                                      &counter_per_kernel,
                                      &offsets_per_kernel,
                                      &out_index,
                                      &unique_key,
                                      &unique_value,
                                      out,
                                      &h_counter,
                                      &offsets);

  const int* counter_ptr = counter_per_kernel.data<int>();
  const int* offsets_ptr = counter_per_kernel.data<int>();

  // 2. gather
  DenseTensorMeta in_features_meta(
      x.dtype(), {n, in_channels}, DataLayout::NCHW);
  DenseTensorMeta out_features_meta(
      x.dtype(), {n, out_channels}, DataLayout::NCHW);
  phi::DenseTensor in_features =
      phi::Empty(dev_ctx, std::move(in_features_meta));
  phi::DenseTensor out_features =
      phi::Empty(dev_ctx, std::move(out_features_meta));
  // T* in_features_ptr = in_features.mutable_data<T>(place);
  dev_ctx.Alloc(
      &in_features, in_features.dtype(), sizeof(T) * in_features.numel());
  T* in_features_ptr = in_features.data<T>();
  // T* out_features_ptr = out_features.mutable_data<T>(place);
  dev_ctx.Alloc(
      &out_features, out_features.dtype(), sizeof(T) * out_features.numel());
  T* out_features_ptr = out_features.data<T>();

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n * in_channels, 1);
  GatherKernel<T, int><<<config.block_per_grid.x,
                         config.thread_per_block.x,
                         0,
                         dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                                             rulebook->data<int>(),
                                             in_features_ptr,
                                             n,
                                             in_channels);

  // 3. call gemm for every werght
  auto blas = phi::funcs::GetBlas<Context, T>(dev_ctx);
  // T* out_values_ptr =
  // out->mutable_non_zero_elements()->mutable_data<T>(place);
  dev_ctx.Alloc(out->mutable_non_zero_elements(),
                out->mutable_non_zero_elements()->dtype(),
                sizeof(T) * in_features.numel());
  T* out_values_ptr = out->mutable_non_zero_elements()->data<T>();
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(out_values_ptr,
                      0,
                      sizeof(T) * out->nnz() * out_channels,
                      dev_ctx.stream()));

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (h_counter[i] <= 0) {
      continue;
    }

    // call gemm: (n, in_channels) * (in_channels, out_channels)
    const int M = h_counter[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + offsets[i] * in_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * K * N;
    T* tmp_out_ptr = out_features_ptr + offsets[i] * out_channels;

    blas.GEMM(CblasNoTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_out_ptr);
  }

  // 4. scatter
  config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, out->nnz() * out_channels, 1);
  ScatterKernel<T><<<config.block_per_grid.x,
                     config.thread_per_block.x,
                     0,
                     dev_ctx.stream()>>>(out_features_ptr,
                                         unique_value.data<int>(),
                                         out_index.data<int>(),
                                         out->nnz(),
                                         n,
                                         out_channels,
                                         out_values_ptr);
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(sparse_conv3d,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
