#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "paddle/phi/api/lib/utils/allocator.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/index_impl.cu.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/primitive/compute_primitives.h"
#include "paddle/phi/kernels/sparse/convolution_kernel.h"
#include "paddle/phi/kernels/sparse/gpu/convolution.cu.h"

namespace phi {
namespace sparse {

__global__ void SetFlagAndUpdateCounterKernel(const int* indexs,
                                              const int n,
                                              const int rulebook_len,
                                              const int kernel_size,
                                              int* rulebook_ptr,
                                              int* counter_ptr) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  extern __shared__ int cache_count[];  // kernel_size
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    cache_count[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < n; i += gridDim.x * blockDim.x) {
    int index = indexs[i];
    int kernel_index = rulebook_ptr[index];
    rulebook_ptr[index + rulebook_len] = -1;
    rulebook_ptr[index + 2 * rulebook_len] = -1;
    rulebook_ptr[index] = -1;
    atomicAdd(&cache_count[kernel_index], 1);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    atomicSub(&counter_ptr[i], cache_count[i]);
  }
}

/**
 * @brief: update the out index and indices
 * unique_keys: save the index of the output feature list
 * unique_values: indiates the index of key before deduplication
 * out_indexs: indicates the position of the output index in the rulebook
 * rulebook_len: indicates the length of rulebook
 * out_dims: indicates the output dims
 * out_indices: the indices of output, out_indices = IndexToPoint(unique_keys)
 * rulebook_out_indexs: the output index in rulebook
**/
__global__ void UpdateIndexKernel(const int* unique_keys,
                                  const int* unique_values,
                                  const int* out_indexs,
                                  const int non_zero_num,
                                  const int rulebook_len,
                                  const Dims4D out_dims,
                                  int* out_indices,
                                  int* rulebook_out_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    const int index = unique_keys[i];
    int batch, x, y, z;
    IndexToPoint<Dims4D>(index, out_dims, &batch, &x, &y, &z);
    // get out indices
    out_indices[i] = batch;
    out_indices[i + non_zero_num] = z;
    out_indices[i + non_zero_num * 2] = y;
    out_indices[i + non_zero_num * 3] = x;

    // update rulebook
    int start = unique_values[i];
    int end = i == non_zero_num - 1 ? rulebook_len : unique_values[i + 1];
    // max(end-start) = kernel_size
    for (int j = start; j < end; j++) {
      rulebook_out_indexs[out_indexs[j]] = i;
    }
  }
}

/**
 * @brief product rulebook
 * for input_i in x_indices:
 *   if input_i participate in the convolution calculation:
 *       infer the output_i by input_i and kernel_i
 *       save output_i
 *
 * x_indices: the indices of input features
 * x_dims: the input dims
 * kernel_dims: the kernel dims
 * out_dims: the output dims
 * non_zero_num: the number of input features
 * rulebook: the rulebook to save the kernel index, input index and output index
 * counter: save the number of times each location in the kernel participates in
 *the caculation
**/
__global__ void ProductRuleBookKernel(const int* x_indices,
                                      const Dims4D x_dims,
                                      const Dims4D kernel_dims,
                                      const Dims4D out_dims,
                                      const int64_t non_zero_num,
                                      const Dims4D paddings,
                                      const Dims4D dilations,
                                      const Dims4D strides,
                                      const bool subm,
                                      int* rulebook,
                                      int* counter,
                                      int* in_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  extern __shared__ int counter_buf[];  // kernel_size
  const int kernel_size = kernel_dims[3] * kernel_dims[2] * kernel_dims[1];
  const int offset = kernel_size * non_zero_num;
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    counter_buf[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int kernel_index = 0;
    int batch = x_indices[i];
    int in_z = x_indices[i + non_zero_num];
    int in_y = x_indices[i + 2 * non_zero_num];
    int in_x = x_indices[i + 3 * non_zero_num];
    if (subm) {
      in_indexs[i] = PointToIndex(batch, in_x, in_y, in_z, x_dims);
    }
    for (int kz = 0; kz < kernel_dims[1]; kz++) {
      for (int ky = 0; ky < kernel_dims[2]; ky++) {
        for (int kx = 0; kx < kernel_dims[3]; kx++) {
          int in_i = -1, out_index = -1, kernel_i = -1;
          if (Check(x_dims,
                    kernel_dims,
                    paddings,
                    dilations,
                    strides,
                    in_x,
                    in_y,
                    in_z,
                    kx,
                    ky,
                    kz)) {
            int out_z = (in_z + paddings[1] - kz * dilations[1]) / strides[1];
            int out_y = (in_y + paddings[2] - ky * dilations[2]) / strides[2];
            int out_x = (in_x + paddings[3] - kx * dilations[3]) / strides[3];
            in_i = i;
            out_index =
                PointToIndex<Dims4D>(batch, out_x, out_y, out_z, out_dims);
            atomicAdd(&counter_buf[kernel_index], 1);
            kernel_i = kernel_index;
          }
          rulebook[kernel_index * non_zero_num + i] = kernel_i;
          rulebook[kernel_index * non_zero_num + offset + i] = in_i;
          rulebook[kernel_index * non_zero_num + offset * 2 + i] = out_index;
          ++kernel_index;
        }
      }
    }
  }
  __syncthreads();
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    atomicAdd(&counter[i], counter_buf[i]);
  }
}

// brief: calculation the distance between start and end
__global__ void DistanceKernel(const int* start,
                               const int* end,
                               int* distance) {
  if (threadIdx.x == 0) {
    *distance = end - start;
  }
}

// the basic algorithm can refer to convolution_kernel.cc or
// the second paper
// example:
// 1. the rulebook:
//  the kernel_index:                       0, 0, 0, 1, 1, 1, 2, 2, ....
//  the out_index(key):                     20, 30, 33, 30, 33, 20, 25
// 2. mark the index of out_index(value):   0, 1, 2, 3, 4, 5, 6, ....
// 3. sorted the (key, value)
// 4. unique the (key, value):
//  unique_key:     20, 25, 30, 33
//  unique_values:  0, 2, 3, 5
//  the index of unique_values is: 0, 1, 2, 3
// 5. update the out_index by unique_key, uniqe_value and the index of
// unique_value:
//  the new out_index: 0, 2, 3, 2, 3, 0, 1
template <typename T, typename Context>
int ProductRuleBook(const Context& dev_ctx,
                    const SparseCooTensor& x,
                    const DenseTensor& kernel,
                    const std::vector<int>& paddings,
                    const std::vector<int>& dilations,
                    const std::vector<int>& strides,
                    const DDim& out_dims,
                    const bool subm,
                    DenseTensor* rulebook,
                    DenseTensor* counter_per_kernel,
                    DenseTensor* offsets_per_kernel,
                    DenseTensor* out_index,
                    DenseTensor* unique_key,
                    DenseTensor* unique_value,
                    SparseCooTensor* out,
                    std::vector<int>* h_counter,
                    std::vector<int>* h_offsets) {
  const auto& kernel_dims = kernel.dims();
  const int64_t non_zero_num = x.nnz();
  const auto& non_zero_indices = x.non_zero_indices();
  const int* indices_ptr = non_zero_indices.data<int>();
  DenseTensor in_indexs = phi::Empty<Context>(
      dev_ctx, DenseTensorMeta(DataType::INT32, {x.nnz()}, DataLayout::NCHW));
  int* counter_ptr = counter_per_kernel->data<int>();
  int* offsets_ptr = offsets_per_kernel->data<int>();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  const int rulebook_rows = 3;
  const int rulebook_cols = kernel_size * non_zero_num;
  rulebook->ResizeAndAllocate({rulebook_rows, rulebook_cols});
  int* rulebook_ptr = rulebook->data<int>();

  const auto x_dims = x.dims();
  Dims4D d_x_dims(x_dims[0], x_dims[3], x_dims[2], x_dims[1]);
  Dims4D d_kernel_dims(1, kernel_dims[2], kernel_dims[1], kernel_dims[0]);
  Dims4D d_out_dims(out_dims[0], out_dims[3], out_dims[2], out_dims[1]);
  Dims4D d_paddings(1, paddings[2], paddings[1], paddings[0]);
  Dims4D d_strides(1, strides[2], strides[1], strides[0]);
  Dims4D d_dilations(1, dilations[2], dilations[1], dilations[0]);

  // 1. product rule book
  phi::funcs::SetConstant<Context, int> set_zero;
  set_zero(dev_ctx, counter_per_kernel, 0);
  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, non_zero_num, 1);

  ProductRuleBookKernel<<<config.block_per_grid.x,
                          config.thread_per_block.x,
                          kernel_size * sizeof(int),
                          dev_ctx.stream()>>>(indices_ptr,
                                              d_x_dims,
                                              d_kernel_dims,
                                              d_out_dims,
                                              non_zero_num,
                                              d_paddings,
                                              d_dilations,
                                              d_strides,
                                              subm,
                                              rulebook_ptr,
                                              counter_ptr,
                                              in_indexs.data<int>());

// 2. remove -1
#ifdef PADDLE_WITH_HIP
  int* last = thrust::remove(thrust::hip::par.on(dev_ctx.stream()),
#else
  int* last = thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                             rulebook_ptr,
                             rulebook_ptr + rulebook_rows * rulebook_cols,
                             -1);

  DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
      rulebook_ptr, last, rulebook_ptr + 3 * kernel_size * non_zero_num - 1);
  int rulebook_len = 0;
  phi::backends::gpu::GpuMemcpyAsync(
      &rulebook_len,
      rulebook_ptr + 3 * kernel_size * non_zero_num - 1,
      sizeof(int),
#ifdef PADDLE_WITH_HIP
      hipMemcpyDeviceToHost,
#else
      hipMemcpyDeviceToHost,
#endif
      dev_ctx.stream());
  rulebook_len /= 3;
  dev_ctx.Wait();

  if (subm) {
    // At present, hashtable is not used to map the input and output indexes.
    // At present, the intermediate output index is generated by normal
    // convolution,
    // and then the intermediate output index is subtracted from the input index
    // to obain the rulebook.
    // get difference
    int32_t* A_key_ptr = rulebook_ptr + 2 * rulebook_len;
    int32_t* B_key_ptr = in_indexs.data<int>();
    DenseTensor A_val = phi::Empty<Context>(
        dev_ctx,
        DenseTensorMeta(DataType::INT32, {rulebook_len}, DataLayout::NCHW));
    DenseTensor B_val = phi::Empty<Context>(
        dev_ctx, DenseTensorMeta(DataType::INT32, {x.nnz()}, DataLayout::NCHW));
    phi::IndexKernel<int, kps::IdentityFunctor<int>>(
        dev_ctx, &A_val, kps::IdentityFunctor<int>());
    phi::IndexKernel<int, kps::IdentityFunctor<int>>(
        dev_ctx, &B_val, kps::IdentityFunctor<int>());
    DenseTensor key_result = phi::Empty<Context>(
        dev_ctx,
        DenseTensorMeta(DataType::INT32, {rulebook_len + 1}, DataLayout::NCHW));
    DenseTensor val_result = phi::Empty<Context>(
        dev_ctx,
        DenseTensorMeta(DataType::INT32, {rulebook_len}, DataLayout::NCHW));

#ifdef PADDLE_WITH_HIP
    thrust::exclusive_scan(thrust::hip::par.on(dev_ctx.stream()),
#else
    thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                           counter_ptr,
                           counter_ptr + kernel_size,
                           offsets_ptr);
    std::vector<int> offsets(kernel_size, 0);
    // TODO(zhangkaihuo): used unified memcpy interface
    phi::backends::gpu::GpuMemcpyAsync(offsets.data(),
                                       offsets_ptr,
                                       kernel_size * sizeof(int),
#ifdef PADDLE_WITH_HIP
                                       hipMemcpyDeviceToHost,
#else
                                       hipMemcpyDeviceToHost,
#endif
                                       dev_ctx.stream());
    dev_ctx.Wait();

    thrust::pair<int*, int*> end;
    // Because set_diff does not support duplicate data, set_diff is performed
    // separately for each segment of data.
    // TODO(zhangkaihuo): Using hashtable here may get better performance,
    // further tests ared needed.
    for (int i = 0; i < kernel_size; i++) {
      int start = offsets[i];
      int stop = i == kernel_size - 1 ? rulebook_len : offsets[i + 1];
      int* key_result_start = (i == 0 ? key_result.data<int>() : end.first);
      int* val_result_start = i == 0 ? val_result.data<int>() : end.second;
      end =
#ifdef PADDLE_WITH_HIP
          thrust::set_difference_by_key(thrust::hip::par.on(dev_ctx.stream()),
#else
          thrust::set_difference_by_key(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                                        A_key_ptr + start,
                                        A_key_ptr + stop,
                                        B_key_ptr,
                                        B_key_ptr + x.nnz(),
                                        A_val.data<int>() + start,
                                        B_val.data<int>(),
                                        key_result_start,
                                        val_result_start);
    }

    DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
        key_result.data<int>(),
        end.first,
        key_result.data<int>() + rulebook_len);
    int len = 0;
    phi::backends::gpu::GpuMemcpyAsync(&len,
                                       key_result.data<int>() + rulebook_len,
                                       sizeof(int),
#ifdef PADDLE_WITH_HIP
                                       hipMemcpyDeviceToHost,
#else
                                       hipMemcpyDeviceToHost,
#endif
                                       dev_ctx.stream());
    dev_ctx.Wait();
    // set the diff value = -1, and update counter
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, len, 1);
    SetFlagAndUpdateCounterKernel<<<config.block_per_grid.x,
                                    config.thread_per_block,
                                    kernel_size * sizeof(int),
                                    dev_ctx.stream()>>>(val_result.data<int>(),
                                                        len,
                                                        rulebook_len,
                                                        kernel_size,
                                                        rulebook_ptr,
                                                        counter_ptr);
// remove -1
#ifdef PADDLE_WITH_HIP
    int* last = thrust::remove(thrust::hip::par.on(dev_ctx.stream()),
#else
    int* last = thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                               rulebook_ptr,
                               rulebook_ptr + 3 * rulebook_len,
                               -1);
    DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
        rulebook_ptr, last, key_result.data<int>() + rulebook_len);
    phi::backends::gpu::GpuMemcpyAsync(&rulebook_len,
                                       key_result.data<int>() + rulebook_len,
                                       sizeof(int),
#ifdef PADDLE_WITH_HIP
                                       hipMemcpyDeviceToHost,
#else
                                       hipMemcpyDeviceToHost,
#endif
                                       dev_ctx.stream());
    dev_ctx.Wait();
    rulebook_len /= 3;
  }

#ifdef PADDLE_WITH_HIP
  thrust::exclusive_scan(thrust::hip::par.on(dev_ctx.stream()),
#else
  thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                         counter_ptr,
                         counter_ptr + kernel_size,
                         offsets_ptr);

#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(&(*h_counter)[0],
                                     counter_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(&(*h_offsets)[0],
                                     offsets_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
#else
  phi::backends::gpu::GpuMemcpyAsync(&(*h_counter)[0],
                                     counter_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(&(*h_offsets)[0],
                                     offsets_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
#endif
  rulebook->Resize({rulebook_rows, rulebook_len});

  // 3. sorted or merge the out index
  out_index->ResizeAndAllocate({rulebook_len});
  unique_value->ResizeAndAllocate({rulebook_len});
  unique_key->ResizeAndAllocate({rulebook_len});
  int* out_index_ptr = out_index->data<int>();
  int* unique_value_ptr = unique_value->data<int>();
  int* unique_key_ptr = unique_key->data<int>();

  int* new_end = SortedAndUniqueIndex(dev_ctx,
                                      rulebook_ptr + 2 * rulebook_len,
                                      rulebook_len,
                                      out_index,
                                      unique_key,
                                      unique_value);
  // thrust::distance doesn't support stream parameters
  // const int out_non_zero_num = thrust::distance(unique_key_ptr,
  // new_end.first);
  DistanceKernel<<<1, 1>>>(unique_key_ptr,
                           new_end,
                           rulebook_ptr + rulebook_rows * rulebook_cols - 1);
  int out_non_zero_num = 0;
#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(
      &out_non_zero_num,
      rulebook_ptr + rulebook_rows * rulebook_cols - 1,
      sizeof(int),
      hipMemcpyDeviceToHost,
      dev_ctx.stream());
#else
  phi::backends::gpu::GpuMemcpyAsync(
      &out_non_zero_num,
      rulebook_ptr + rulebook_rows * rulebook_cols - 1,
      sizeof(int),
      hipMemcpyDeviceToHost,
      dev_ctx.stream());
#endif
  dev_ctx.Wait();

  // 5. update out_indices and rulebook by unique_value_ptr
  const int64_t sparse_dim = 4;
  DenseTensorMeta indices_meta(
      DataType::INT32, {sparse_dim, out_non_zero_num}, DataLayout::NCHW);
  DenseTensorMeta values_meta(
      x.dtype(), {out_non_zero_num, kernel_dims[4]}, x.layout());
  phi::DenseTensor out_indices = phi::Empty(dev_ctx, std::move(indices_meta));
  phi::DenseTensor out_values = phi::Empty(dev_ctx, std::move(values_meta));

  int* out_indices_ptr = out_indices.data<int>();

  config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, out_non_zero_num, 1);
  UpdateIndexKernel<<<config.block_per_grid.x,
                      config.thread_per_block.x,
                      0,
                      dev_ctx.stream()>>>(unique_key_ptr,
                                          unique_value_ptr,
                                          out_index_ptr,
                                          out_non_zero_num,
                                          rulebook_len,
                                          d_out_dims,
                                          out_indices_ptr,
                                          rulebook_ptr + 2 * rulebook_len);
  out->SetMember(out_indices, out_values, out_dims, true);
  return rulebook_len;
}

/**
 * x: (N, D, H, W, C)
 * kernel: (D, H, W, C, OC)
 * out: (N, D, H, W, OC)
**/
template <typename T, typename Context>
void Conv3dKernel(const Context& dev_ctx,
                  const SparseCooTensor& x,
                  const DenseTensor& kernel,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const int groups,
                  const bool subm,
                  SparseCooTensor* out,
                  DenseTensor* rulebook) {
  // update padding and dilation
  // Currently, only support x.layout is NDHWC, groups = 1
  // if x.layout != NDHWC then transpose(x), transpose(weight)

  const auto& x_dims = x.dims();
  const auto& kernel_dims = kernel.dims();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  DDim out_dims = {1, 1, 1, 1, 1};
  GetOutShape(x_dims, kernel_dims, paddings, dilations, strides, &out_dims);
  out->set_dims(out_dims);
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];
  std::vector<int> offsets(kernel_size + 1), h_counter(kernel_size);

  // Second algorithm:
  // https://pdfs.semanticscholar.org/5125/a16039cabc6320c908a4764f32596e018ad3.pdf
  // 1. product rulebook
  DenseTensorMeta counter_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensorMeta offsets_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensor counter_per_kernel = phi::Empty(dev_ctx, std::move(counter_meta));
  DenseTensor offsets_per_kernel = phi::Empty(dev_ctx, std::move(offsets_meta));
  DenseTensorMeta index_meta(DataType::INT32, {1}, DataLayout::NCHW);
  DenseTensor out_index = phi::Empty(dev_ctx, std::move(index_meta));
  DenseTensor unique_key = phi::Empty(dev_ctx, std::move(index_meta));
  DenseTensor unique_value = phi::Empty(dev_ctx, std::move(index_meta));

  std::vector<int> subm_paddings(paddings), subm_strides(strides);
  if (subm) {
    auto kernel_dims = kernel.dims();
    for (int i = 0; i < paddings.size(); i++) {
      subm_paddings[i] = kernel_dims[i] / 2;
      subm_strides[i] = 1;
    }
  }

  int n = ProductRuleBook<T, Context>(dev_ctx,
                                      x,
                                      kernel,
                                      subm_paddings,
                                      dilations,
                                      subm_strides,
                                      out_dims,
                                      subm,
                                      rulebook,
                                      &counter_per_kernel,
                                      &offsets_per_kernel,
                                      &out_index,
                                      &unique_key,
                                      &unique_value,
                                      out,
                                      &h_counter,
                                      &offsets);

  const int* counter_ptr = counter_per_kernel.data<int>();
  const int* offsets_ptr = counter_per_kernel.data<int>();
  const int* rulebook_ptr = rulebook->data<int>();

  // 2. gather
  DenseTensorMeta in_features_meta(
      x.dtype(), {n, in_channels}, DataLayout::NCHW);
  DenseTensorMeta out_features_meta(
      x.dtype(), {n, out_channels}, DataLayout::NCHW);
  phi::DenseTensor in_features =
      phi::Empty(dev_ctx, std::move(in_features_meta));
  phi::DenseTensor out_features =
      phi::Empty(dev_ctx, std::move(out_features_meta));
  T* in_features_ptr = in_features.data<T>();
  T* out_features_ptr = out_features.data<T>();
  phi::funcs::SetConstant<Context, T> set_zero;
  set_zero(dev_ctx, &out_features, static_cast<T>(0.0f));

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n * in_channels, 1);
  GatherKernel<T, int><<<config.block_per_grid.x,
                         config.thread_per_block.x,
                         0,
                         dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                                             rulebook_ptr + n,
                                             in_features_ptr,
                                             n,
                                             in_channels);

  // 3. call gemm for every werght
  auto blas = phi::funcs::GetBlas<Context, T>(dev_ctx);
  auto* out_values = out->mutable_non_zero_elements();
  T* out_values_ptr = out_values->data<T>();

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (h_counter[i] <= 0) {
      continue;
    }

    // call gemm: (n, in_channels) * (in_channels, out_channels)
    const int M = h_counter[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + offsets[i] * in_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * K * N;
    T* tmp_out_ptr = out_features_ptr + offsets[i] * out_channels;

    blas.GEMM(CblasNoTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_out_ptr);
  }

  // 4. scatter
  config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, out->nnz() * out_channels, 1);
  ScatterKernel<T><<<config.block_per_grid.x,
                     config.thread_per_block.x,
                     0,
                     dev_ctx.stream()>>>(out_features_ptr,
                                         unique_value.data<int>(),
                                         out_index.data<int>(),
                                         out->nnz(),
                                         n,
                                         out_channels,
                                         out_values_ptr);
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(sparse_conv3d,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
