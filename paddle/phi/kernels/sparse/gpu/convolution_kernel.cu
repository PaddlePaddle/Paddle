#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "glog/logging.h"
#include "paddle/phi/api/lib/utils/allocator.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/index_impl.cu.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/primitive/compute_primitives.h"
#include "paddle/phi/kernels/sparse/convolution_kernel.h"

namespace phi {
namespace sparse {

// TODO(zhangkaihuo) replace this kernel with KP::InitWithDataIndex
__global__ void InitByIndexKernel(const int n, int* out1, int* out2) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += gridDim.x * blockDim.x) {
    out1[i] = i;
    out2[i] = i;
  }
}

__global__ void SetFlagAndUpdateCounterKernel(const int* indexs,
                                              const int n,
                                              const int rulebook_len,
                                              const int kernel_size,
                                              int* rulebook_ptr,
                                              int* counter_ptr) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  extern __shared__ int cache_count[];  // kernel_size
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    cache_count[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < n; i += gridDim.x * blockDim.x) {
    int index = indexs[i];
    int kernel_index = rulebook_ptr[index];
    rulebook_ptr[index + rulebook_len] = -1;
    rulebook_ptr[index + 2 * rulebook_len] = -1;
    rulebook_ptr[index] = -1;
    atomicAdd(&cache_count[kernel_index], 1);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    atomicSub(&counter_ptr[i], cache_count[i]);
  }
}

/**
 * @brief: update the out index and indices
 * unique_keys: save the index of the output feature list
 * unique_values: indiates the index of key before deduplication
 * out_indexs: indicates the position of the output index in the rulebook
 * rulebook_len: indicates the length of rulebook
 * out_dims: indicates the output dims
 * out_indices: the indices of output, out_indices = IndexToPoint(unique_keys)
 * rulebook_out_indexs: the output index in rulebook
**/
__global__ void UpdateIndexKernel(const int* unique_keys,
                                  const int* unique_values,
                                  const int* out_indexs,
                                  const int non_zero_num,
                                  const int rulebook_len,
                                  const Dims4D out_dims,
                                  int* out_indices,
                                  int* rulebook_out_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    const int index = unique_keys[i];
    int batch, x, y, z;
    IndexToPoint<Dims4D>(index, out_dims, &batch, &x, &y, &z);
    // get out indices
    out_indices[i] = batch;
    out_indices[i + non_zero_num] = z;
    out_indices[i + non_zero_num * 2] = y;
    out_indices[i + non_zero_num * 3] = x;

    // update rulebook
    int start = unique_values[i];
    int end = i == non_zero_num - 1 ? rulebook_len : unique_values[i + 1];
    // max(end-start) = kernel_size
    for (int j = start; j < end; j++) {
      rulebook_out_indexs[out_indexs[j]] = i;
    }
  }
}

/**
 * @brief product rulebook
 * for input_i in x_indices:
 *   if input_i participate in the convolution calculation:
 *       infer the output_i by input_i and kernel_i
 *       save output_i
 *
 * x_indices: the indices of input features
 * x_dims: the input dims
 * kernel_dims: the kernel dims
 * out_dims: the output dims
 * non_zero_num: the number of input features
 * rulebook: the rulebook to save the kernel index, input index and output index
 * counter: save the number of times each location in the kernel participates in
 *the caculation
**/
__global__ void ProductRuleBookKernel(const int* x_indices,
                                      const Dims4D x_dims,
                                      const Dims4D kernel_dims,
                                      const Dims4D out_dims,
                                      const int64_t non_zero_num,
                                      const Dims4D paddings,
                                      const Dims4D dilations,
                                      const Dims4D strides,
                                      const bool subm,
                                      int* rulebook,
                                      int* counter,
                                      int* in_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  extern __shared__ int counter_buf[];  // kernel_size
  const int kernel_size = kernel_dims[3] * kernel_dims[2] * kernel_dims[1];
  const int offset = kernel_size * non_zero_num;
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    counter_buf[i] = 0;
  }
  __syncthreads();

  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int kernel_index = 0;
    int batch = x_indices[i];
    int in_z = x_indices[i + non_zero_num];
    int in_y = x_indices[i + 2 * non_zero_num];
    int in_x = x_indices[i + 3 * non_zero_num];
    if (subm) {
      in_indexs[i] = PointToIndex(batch, in_x, in_y, in_z, x_dims);
    }
    for (int kz = 0; kz < kernel_dims[1]; kz++) {
      for (int ky = 0; ky < kernel_dims[2]; ky++) {
        for (int kx = 0; kx < kernel_dims[3]; kx++) {
          int in_i = -1, out_index = -1, kernel_i = -1;
          if (Check(x_dims,
                    kernel_dims,
                    paddings,
                    dilations,
                    strides,
                    in_x,
                    in_y,
                    in_z,
                    kx,
                    ky,
                    kz)) {
            int out_z = (in_z + paddings[1] - kz * dilations[1]) / strides[1];
            int out_y = (in_y + paddings[2] - ky * dilations[2]) / strides[2];
            int out_x = (in_x + paddings[3] - kx * dilations[3]) / strides[3];
            in_i = i;
            out_index =
                PointToIndex<Dims4D>(batch, out_x, out_y, out_z, out_dims);
            atomicAdd(&counter_buf[kernel_index], 1);
            kernel_i = kernel_index;
          }
          rulebook[kernel_index * non_zero_num + i] = kernel_i;
          rulebook[kernel_index * non_zero_num + offset + i] = in_i;
          rulebook[kernel_index * non_zero_num + 2 * offset + i] = out_index;
          ++kernel_index;
        }
      }
    }
  }
  __syncthreads();
  for (int i = threadIdx.x; i < kernel_size; i += blockDim.x) {
    atomicAdd(&counter[i], counter_buf[i]);
  }
}

// TODO(zhangkaihuo): After the GatherCUDAKernel is migrated to phi, replace
// this kernel with phi::GatherCUDAKernel;
// Vectorization can be used to improve read and write bandwidth
/**
 * brief: gather data from params according to indices
 * params: the inputs
 * indices: the indices you want to gather
 * output: the outputs
 * index_size: the size of indices
 * slice_size: slice size corresponding to each index, here is the channel size
**/
template <typename T, typename IndexT = int>
__global__ void GatherKernel(const T* params,
                             const IndexT* indices,
                             T* output,
                             size_t index_size,
                             size_t slice_size) {
  CUDA_KERNEL_LOOP_TYPE(i, index_size * slice_size, int64_t) {
    int64_t indices_i = i / slice_size;
    int64_t slice_i = i - indices_i * slice_size;  // offset inside the slice
    IndexT gather_i = indices[indices_i];
    int64_t params_i = gather_i * slice_size + slice_i;
    *(output + i) = *(params + params_i);
  }
}

/**
 * brief: scatter add
 * input: the inputs
 * unique_value: refer to UpdateIndexKernel notes
 * out_index: the output feature index
 * non_zero_num: the number of output features
 * rulebook_len: the length of rulebook
 * channels: the output channel size
 * out: the outputs
**/
template <typename T>
__global__ void ScatterKernel(const T* input,
                              const int* unique_value,
                              const int* out_index,
                              const int non_zero_num,
                              const int rulebook_len,
                              const int channels,
                              T* out) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num * channels; i += gridDim.x * blockDim.x) {
    int indices_i = i / channels;
    int channels_i = i - indices_i * channels;

    int start = unique_value[indices_i];
    int end = indices_i == non_zero_num - 1 ? rulebook_len
                                            : unique_value[indices_i + 1];
    // max(end-start) = kernel_size
    T sum = static_cast<T>(0);
    for (int j = start; j < end; j++) {
      const int out_feature_i = out_index[j];
      sum += input[out_feature_i * channels + channels_i];
    }
    out[indices_i * channels + channels_i] = sum;
  }
}

// brief: calculation the distance between start and end
__global__ void DistanceKernel(const int* start,
                               const int* end,
                               int* distance) {
  if (threadIdx.x == 0) {
    *distance = end - start;
  }
}

// the basic algorithm can refer to convolution_kernel.cc or
// the second paper
// example:
// 1. the rulebook:
//  the kernel_index:                       0, 0, 0, 1, 1, 1, 2, 2, ....
//  the out_index(key):                     20, 30, 33, 30, 33, 20, 25
// 2. mark the index of out_index(value):   0, 1, 2, 3, 4, 5, 6, ....
// 3. sorted the (key, value)
// 4. unique the (key, value):
//  unique_key:     20, 25, 30, 33
//  unique_values:  0, 2, 3, 5
//  the index of unique_values is: 0, 1, 2, 3
// 5. update the out_index by unique_key, uniqe_value and the index of
// unique_value:
//  the new out_index: 0, 2, 3, 2, 3, 0, 1
template <typename T, typename Context>
int ProductRuleBook(const Context& dev_ctx,
                    const SparseCooTensor& x,
                    const DenseTensor& kernel,
                    const std::vector<int>& paddings,
                    const std::vector<int>& dilations,
                    const std::vector<int>& strides,
                    const DDim& out_dims,
                    const bool subm,
                    DenseTensor* rulebook,
                    DenseTensor* counter_per_kernel,
                    DenseTensor* offsets_per_kernel,
                    DenseTensor* out_index,
                    DenseTensor* unique_key,
                    DenseTensor* unique_value,
                    SparseCooTensor* out,
                    std::vector<int>* h_counter,
                    std::vector<int>* h_offsets) {
  const auto& kernel_dims = kernel.dims();
  const int64_t non_zero_num = x.nnz();
  const auto& non_zero_indices = x.non_zero_indices();
  const int* indices_ptr = non_zero_indices.data<int>();
  DenseTensor in_indexs = phi::Empty<Context>(
      dev_ctx, DenseTensorMeta(DataType::INT32, {x.nnz()}, DataLayout::NCHW));
  dev_ctx.Alloc(counter_per_kernel,
                counter_per_kernel->dtype(),
                sizeof(int) * counter_per_kernel->numel());
  int* counter_ptr = counter_per_kernel->data<int>();
  dev_ctx.Alloc(offsets_per_kernel,
                offsets_per_kernel->dtype(),
                sizeof(int) * offsets_per_kernel->numel());
  int* offsets_ptr = offsets_per_kernel->data<int>();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  rulebook->ResizeAndAllocate({3, kernel_size * non_zero_num});
  dev_ctx.Alloc(rulebook, rulebook->dtype(), sizeof(int) * rulebook->numel());
  int* rulebook_ptr = rulebook->data<int>();

  const auto x_dims = x.dims();
  Dims4D d_x_dims(x_dims[0], x_dims[3], x_dims[2], x_dims[1]);
  Dims4D d_kernel_dims(1, kernel_dims[2], kernel_dims[1], kernel_dims[0]);
  Dims4D d_out_dims(out_dims[0], out_dims[3], out_dims[2], out_dims[1]);
  Dims4D d_paddings(1, paddings[2], paddings[1], paddings[0]);
  Dims4D d_strides(1, strides[2], strides[1], strides[0]);
  Dims4D d_dilations(1, dilations[2], dilations[1], dilations[0]);

  // 1. product rule book
  phi::funcs::SetConstant<Context, int> set_zero;
  set_zero(dev_ctx, counter_per_kernel, 0);
  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, non_zero_num, 1);

  ProductRuleBookKernel<<<config.block_per_grid.x,
                          config.thread_per_block.x,
                          kernel_size * sizeof(int),
                          dev_ctx.stream()>>>(indices_ptr,
                                              d_x_dims,
                                              d_kernel_dims,
                                              d_out_dims,
                                              non_zero_num,
                                              d_paddings,
                                              d_dilations,
                                              d_strides,
                                              subm,
                                              rulebook_ptr,
                                              counter_ptr,
                                              in_indexs.data<int>());

// 2. remove -1
#ifdef PADDLE_WITH_HIP
  int* last = thrust::remove(thrust::hip::par.on(dev_ctx.stream()),
#else
  int* last = thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                             rulebook_ptr,
                             rulebook_ptr + 3 * kernel_size * non_zero_num,
                             -1);

  DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
      rulebook_ptr, last, rulebook_ptr + 3 * kernel_size * non_zero_num - 1);
  int rulebook_len = 0;
  hipMemcpyAsync(&rulebook_len,
                  rulebook_ptr + 3 * kernel_size * non_zero_num - 1,
                  sizeof(int),
                  hipMemcpyDeviceToHost,
                  dev_ctx.stream());
  rulebook_len /= 3;
  dev_ctx.Wait();

  if (subm) {
    // At present, hashtable is not used to map the input and output indexes.
    // At present, the intermediate output index is generated by normal
    // convolution,
    // and then the intermediate output index is subtracted from the input index
    // to obain the rulebook.
    // get difference
    int32_t* A_key_ptr = rulebook_ptr + 2 * rulebook_len;
    int32_t* B_key_ptr = in_indexs.data<int>();
    DenseTensor A_val = phi::Empty<Context>(
        dev_ctx,
        DenseTensorMeta(DataType::INT32, {rulebook_len}, DataLayout::NCHW));
    DenseTensor B_val = phi::Empty<Context>(
        dev_ctx, DenseTensorMeta(DataType::INT32, {x.nnz()}, DataLayout::NCHW));
    phi::IndexKernel<int, kps::IdentityFunctor<int>>(
        dev_ctx, &A_val, kps::IdentityFunctor<int>());
    phi::IndexKernel<int, kps::IdentityFunctor<int>>(
        dev_ctx, &B_val, kps::IdentityFunctor<int>());
    DenseTensor key_result = phi::Empty<Context>(
        dev_ctx,
        DenseTensorMeta(DataType::INT32, {rulebook_len + 1}, DataLayout::NCHW));
    DenseTensor val_result = phi::Empty<Context>(
        dev_ctx,
        DenseTensorMeta(DataType::INT32, {rulebook_len}, DataLayout::NCHW));

    thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
                           counter_ptr,
                           counter_ptr + kernel_size,
                           offsets_ptr);
    std::vector<int> offsets(kernel_size, 0);
    hipMemcpy(offsets.data(),
               offsets_ptr,
               kernel_size * sizeof(int),
               hipMemcpyDeviceToHost);

    thrust::pair<int*, int*> end;
    // Because set_diff does not support duplicate data, set_diff is performed
    // separately for each segment of data.
    // TODO(zhangkaihuo): Using hashtable here may get better performance,
    // further tests ared needed.
    for (int i = 0; i < kernel_size; i++) {
      int start = offsets[i];
      int stop = i == kernel_size - 1 ? rulebook_len : offsets[i + 1];
      int* key_result_start = (i == 0 ? key_result.data<int>() : end.first);
      int* val_result_start = i == 0 ? val_result.data<int>() : end.second;
      end =
          thrust::set_difference_by_key(thrust::cuda::par.on(dev_ctx.stream()),
                                        A_key_ptr + start,
                                        A_key_ptr + stop,
                                        B_key_ptr,
                                        B_key_ptr + x.nnz(),
                                        A_val.data<int>() + start,
                                        B_val.data<int>(),
                                        key_result_start,
                                        val_result_start);
    }

    DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
        key_result.data<int>(),
        end.first,
        key_result.data<int>() + rulebook_len);
    int len = 0;
    hipMemcpyAsync(&len,
                    key_result.data<int>() + rulebook_len,
                    sizeof(int),
                    hipMemcpyDeviceToHost,
                    dev_ctx.stream());
    dev_ctx.Wait();
    // set the diff value = -1, and update counter
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, len, 1);
    SetFlagAndUpdateCounterKernel<<<config.block_per_grid.x,
                                    config.thread_per_block,
                                    kernel_size * sizeof(int),
                                    dev_ctx.stream()>>>(val_result.data<int>(),
                                                        len,
                                                        rulebook_len,
                                                        kernel_size,
                                                        rulebook_ptr,
                                                        counter_ptr);
    // remove -1
    int* last = thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
                               rulebook_ptr,
                               rulebook_ptr + 3 * rulebook_len,
                               -1);
    DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
        rulebook_ptr, last, key_result.data<int>() + rulebook_len);
    hipMemcpyAsync(&rulebook_len,
                    key_result.data<int>() + rulebook_len,
                    sizeof(int),
                    hipMemcpyDeviceToHost,
                    dev_ctx.stream());
    dev_ctx.Wait();
    rulebook_len /= 3;
  }

#ifdef PADDLE_WITH_HIP
  thrust::exclusive_scan(thrust::hip::par.on(dev_ctx.stream()),
#else
  thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                         counter_ptr,
                         counter_ptr + kernel_size,
                         offsets_ptr);

#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(&(*h_counter)[0],
                                     counter_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(&(*h_offsets)[0],
                                     offsets_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
#else
  phi::backends::gpu::GpuMemcpyAsync(&(*h_counter)[0],
                                     counter_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(&(*h_offsets)[0],
                                     offsets_ptr,
                                     kernel_size * sizeof(int),
                                     hipMemcpyDeviceToHost,
                                     dev_ctx.stream());
#endif

  // 3. sorted or merge the out index
  out_index->ResizeAndAllocate({rulebook_len});
  unique_value->ResizeAndAllocate({rulebook_len});
  unique_key->ResizeAndAllocate({rulebook_len});
  dev_ctx.Alloc(
      out_index, out_index->dtype(), sizeof(int) * out_index->numel());
  int* out_index_ptr = out_index->data<int>();
  dev_ctx.Alloc(
      unique_value, unique_value->dtype(), sizeof(int) * unique_value->numel());
  int* unique_value_ptr = unique_value->data<int>();
  dev_ctx.Alloc(
      unique_key, unique_key->dtype(), sizeof(int) * unique_key->numel());
  int* unique_key_ptr = unique_key->data<int>();

  config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, rulebook_len, 1);
  InitByIndexKernel<<<config.block_per_grid.x,
                      config.thread_per_block.x,
                      0,
                      dev_ctx.stream()>>>(
      rulebook_len, out_index_ptr, unique_value_ptr);

#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(unique_key_ptr,
                                     rulebook_ptr + 2 * rulebook_len,
                                     rulebook_len * sizeof(int),
                                     hipMemcpyDeviceToDevice,
                                     dev_ctx.stream());
#else
  phi::backends::gpu::GpuMemcpyAsync(unique_key_ptr,
                                     rulebook_ptr + 2 * rulebook_len,
                                     rulebook_len * sizeof(int),
                                     hipMemcpyDeviceToDevice,
                                     dev_ctx.stream());
#endif

// compared with thrust::sort_by_key, thrust::merge_by_key may achieved higher
// performance, but thrust::merge_by_key limited by data size
#ifdef PADDLE_WITH_HIP
  thrust::sort_by_key(thrust::hip::par.on(dev_ctx.stream()),
#else
  thrust::sort_by_key(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                      unique_key_ptr,
                      unique_key_ptr + rulebook_len,
                      out_index_ptr);

  // 4. unique
  thrust::pair<int*, int*> new_end =
#ifdef PADDLE_WITH_HIP
      thrust::unique_by_key(thrust::hip::par.on(dev_ctx.stream()),
#else
      thrust::unique_by_key(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                            unique_key_ptr,
                            unique_key_ptr + rulebook_len,
                            unique_value_ptr);
  // thrust::distance doesn't support stream parameters
  // const int out_non_zero_num = thrust::distance(unique_key_ptr,
  // new_end.first);
  DistanceKernel<<<1, 1, 0, dev_ctx.stream()>>>(
      unique_key_ptr,
      new_end.first,
      rulebook_ptr + 3 * kernel_size * non_zero_num - 1);
  int out_non_zero_num = 0;
#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(
      &out_non_zero_num,
      rulebook_ptr + 3 * kernel_size * non_zero_num - 1,
      sizeof(int),
      hipMemcpyDeviceToHost,
      dev_ctx.stream());
#else
  phi::backends::gpu::GpuMemcpyAsync(
      &out_non_zero_num,
      rulebook_ptr + 3 * kernel_size * non_zero_num - 1,
      sizeof(int),
      hipMemcpyDeviceToHost,
      dev_ctx.stream());
#endif
  dev_ctx.Wait();

  // 5. update out_indices and rulebook by unique_value_ptr
  const int64_t sparse_dim = 4;
  DenseTensorMeta indices_meta(
      DataType::INT32, {sparse_dim, out_non_zero_num}, DataLayout::NCHW);
  DenseTensorMeta values_meta(
      x.dtype(), {out_non_zero_num, kernel_dims[4]}, x.layout());
  phi::DenseTensor out_indices = phi::Empty(dev_ctx, std::move(indices_meta));
  phi::DenseTensor out_values = phi::Empty(dev_ctx, std::move(values_meta));

  dev_ctx.Alloc(
      &out_indices, out_indices.dtype(), sizeof(int) * out_indices.numel());
  int* out_indices_ptr = out_indices.data<int>();

  config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, out_non_zero_num, 1);
  UpdateIndexKernel<<<config.block_per_grid.x,
                      config.thread_per_block.x,
                      0,
                      dev_ctx.stream()>>>(unique_key_ptr,
                                          unique_value_ptr,
                                          out_index_ptr,
                                          out_non_zero_num,
                                          rulebook_len,
                                          d_out_dims,
                                          out_indices_ptr,
                                          rulebook_ptr + 2 * rulebook_len);
  out->SetMember(out_indices, out_values, out_dims, true);
  return rulebook_len;
}

/**
 * x: (N, D, H, W, C)
 * kernel: (D, H, W, C, OC)
 * out: (N, D, H, W, OC)
**/
template <typename T, typename Context>
void Conv3dKernel(const Context& dev_ctx,
                  const SparseCooTensor& x,
                  const DenseTensor& kernel,
                  const std::vector<int>& paddings,
                  const std::vector<int>& dilations,
                  const std::vector<int>& strides,
                  const int groups,
                  const bool subm,
                  SparseCooTensor* out,
                  DenseTensor* rulebook) {
  // update padding and dilation
  // Currently, only support x.layout is NDHWC, groups = 1
  // if x.layout != NDHWC then transpose(x), transpose(weight)

  const auto& x_dims = x.dims();
  const auto& kernel_dims = kernel.dims();
  int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  DDim out_dims = {1, 1, 1, 1, 1};
  GetOutShape(x_dims, kernel_dims, paddings, dilations, strides, &out_dims);
  out->set_dims(out_dims);
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];
  std::vector<int> offsets(kernel_size + 1), h_counter(kernel_size);

  // Second algorithm:
  // https://pdfs.semanticscholar.org/5125/a16039cabc6320c908a4764f32596e018ad3.pdf
  // 1. product rulebook
  DenseTensorMeta counter_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensorMeta offsets_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensor counter_per_kernel = phi::Empty(dev_ctx, std::move(counter_meta));
  DenseTensor offsets_per_kernel = phi::Empty(dev_ctx, std::move(offsets_meta));
  DenseTensor out_index = phi::Empty<int, Context>(dev_ctx);
  DenseTensor unique_key = phi::Empty<int, Context>(dev_ctx);
  DenseTensor unique_value = phi::Empty<int, Context>(dev_ctx);

  int n = ProductRuleBook<T, Context>(dev_ctx,
                                      x,
                                      kernel,
                                      paddings,
                                      dilations,
                                      strides,
                                      out_dims,
                                      subm,
                                      rulebook,
                                      &counter_per_kernel,
                                      &offsets_per_kernel,
                                      &out_index,
                                      &unique_key,
                                      &unique_value,
                                      out,
                                      &h_counter,
                                      &offsets);

  const int* counter_ptr = counter_per_kernel.data<int>();
  const int* offsets_ptr = counter_per_kernel.data<int>();

  // 2. gather
  DenseTensorMeta in_features_meta(
      x.dtype(), {n, in_channels}, DataLayout::NCHW);
  DenseTensorMeta out_features_meta(
      x.dtype(), {n, out_channels}, DataLayout::NCHW);
  phi::DenseTensor in_features =
      phi::Empty(dev_ctx, std::move(in_features_meta));
  phi::DenseTensor out_features =
      phi::Empty(dev_ctx, std::move(out_features_meta));
  dev_ctx.Alloc(
      &in_features, in_features.dtype(), sizeof(T) * in_features.numel());
  T* in_features_ptr = in_features.data<T>();
  dev_ctx.Alloc(
      &out_features, out_features.dtype(), sizeof(T) * out_features.numel());
  T* out_features_ptr = out_features.data<T>();
  phi::funcs::SetConstant<Context, T> set_zero;
  set_zero(dev_ctx, &out_features, static_cast<T>(0.0f));

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n * in_channels, 1);
  GatherKernel<T, int><<<config.block_per_grid.x,
                         config.thread_per_block.x,
                         0,
                         dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                                             rulebook->data<int>() + n,
                                             in_features_ptr,
                                             n,
                                             in_channels);

  // 3. call gemm for every werght
  auto blas = phi::funcs::GetBlas<Context, T>(dev_ctx);
  auto* out_values = out->mutable_non_zero_elements();
  dev_ctx.Alloc(
      out_values, out_values->dtype(), sizeof(T) * out_values->numel());
  T* out_values_ptr = out_values->data<T>();

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (h_counter[i] <= 0) {
      continue;
    }

    // call gemm: (n, in_channels) * (in_channels, out_channels)
    const int M = h_counter[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + offsets[i] * in_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * K * N;
    T* tmp_out_ptr = out_features_ptr + offsets[i] * out_channels;

    blas.GEMM(CblasNoTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_out_ptr);
  }

  // 4. scatter
  config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, out->nnz() * out_channels, 1);
  ScatterKernel<T><<<config.block_per_grid.x,
                     config.thread_per_block.x,
                     0,
                     dev_ctx.stream()>>>(out_features_ptr,
                                         unique_value.data<int>(),
                                         out_index.data<int>(),
                                         out->nnz(),
                                         n,
                                         out_channels,
                                         out_values_ptr);
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(sparse_conv3d,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
