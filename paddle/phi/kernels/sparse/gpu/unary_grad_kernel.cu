#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/sparse/unary_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/sparse/impl/unary_grad_kernel_impl.h"

#define PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(name, prefix)     \
  PD_REGISTER_KERNEL(name##_coo_grad,                              \
                     GPU,                                          \
                     ALL_LAYOUT,                                   \
                     phi::sparse::prefix##CooGradKernel,           \
                     phi::dtype::float16,                          \
                     float,                                        \
                     double) {                                     \
    kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO); \
  }                                                                \
                                                                   \
  PD_REGISTER_KERNEL(name##_csr_grad,                              \
                     GPU,                                          \
                     ALL_LAYOUT,                                   \
                     phi::sparse::prefix##CsrGradKernel,           \
                     phi::dtype::float16,                          \
                     float,                                        \
                     double) {                                     \
    kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR); \
  }

PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(sin, Sin)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(tan, Tan)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(asin, Asin)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(atan, Atan)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(sinh, Sinh)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(tanh, Tanh)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(asinh, Asinh)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(atanh, Atanh)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(sqrt, Sqrt)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(square, Square)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(log1p, Log1p)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(relu, Relu)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(abs, Abs)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(pow, Pow)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(expm1, Expm1)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(relu6, Relu6)
PD_REGISTER_SPARSE_UNARY_GPU_GRAD_KERNEL(leaky_relu, LeakyRelu)

PD_REGISTER_KERNEL(cast_coo_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CastCooGradKernel,
                   phi::dtype::float16,
                   float,
                   double,
                   int8_t,
                   uint8_t,
                   int16_t,
                   int,
                   int64_t,
                   bool) {}

PD_REGISTER_KERNEL(cast_csr_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CastCsrGradKernel,
                   phi::dtype::float16,
                   float,
                   double,
                   int8_t,
                   uint8_t,
                   int16_t,
                   int,
                   int64_t,
                   bool) {}
