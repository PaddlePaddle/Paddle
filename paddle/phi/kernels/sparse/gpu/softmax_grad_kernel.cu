#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"
#include "paddle/phi/kernels/sparse/softmax_grad_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT = int>
__global__ void SoftmaxGradGpuKernel(const IntT* out_crows,
                                     const T* out_values,
                                     const T* dout_values,
                                     T* dx_values,
                                     int row_number) {
  // dx = (dout - sum(dout * out)) * out
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  int non_zero_idx = threadIdx.x;
  if (row >= row_number) return;
  int row_first = static_cast<int>(out_crows[row]);
  int row_nnz = static_cast<int>(out_crows[row + 1] - out_crows[row]);
  if (row_nnz == 0) return;

  int kIteration = (row_nnz + warpSize - 1) / warpSize;

  T mul_result = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    mul_result += out_values[row_first + idx] * dout_values[row_first + idx];
  }
  T sum = phi::funcs::warpReduceSum<T>(mul_result, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    dx_values[row_first + idx] =
        (dout_values[row_first + idx] - sum) * out_values[row_first + idx];
  }
}

template <typename T, typename Context>
void SoftmaxCsrGradKernel(const Context& dev_ctx,
                          const SparseCsrTensor& out,
                          const SparseCsrTensor& dout,
                          int axis,
                          SparseCsrTensor* dx) {
  PADDLE_ENFORCE_EQ(axis,
                    -1,
                    phi::errors::Unimplemented(
                        "SparseCsrTensor only support axis=-1 for softmax, "
                        "which is faster when reading data by row (axis=-1)"));
  EmptyLikeCsrKernel<T, Context>(dev_ctx, dout, dx);

  auto out_dim = out.dims();
  int row_number = 1;
  for (int i = 0; i < out_dim.size() - 1; ++i) {
    row_number *= out_dim[i];
  }

  dim3 grid((row_number + 3) / 4);
  dim3 block(32, 4);

  PD_VISIT_INTEGRAL_TYPES(
      out.non_zero_crows().dtype(), "SoftmaxCsrGradKernel", ([&] {
        SoftmaxGradGpuKernel<T, data_t><<<grid, block, 0, dev_ctx.stream()>>>(
            out.non_zero_crows().data<data_t>(),
            out.non_zero_elements().data<T>(),
            dout.non_zero_elements().data<T>(),
            dx->mutable_non_zero_elements()->data<T>(),
            row_number);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(softmax_csr_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCsrGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}
