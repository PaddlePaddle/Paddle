#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/softmax_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT = int>
__global__ void SoftmaxGradGpuKernel(const IntT* out_crows,
                                     const T* out_values,
                                     const T* dout_values,
                                     T* dx_values,
                                     int row_number,
                                     int total_row_number) {
  // dx = (dout - sum(dout * out)) * out
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  int non_zero_idx = threadIdx.x;
  if (row >= total_row_number) return;
  int cur_batch = row / row_number;
  int crow_idx = cur_batch * (row_number + 1) + (row % row_number);
  int cur_batch_offset = 0;
  for (int i = 1; i < cur_batch + 1; ++i) {
    cur_batch_offset += out_crows[i * (row_number + 1) - 1];
  }
  int row_first = cur_batch_offset + static_cast<int>(out_crows[crow_idx]);
  int row_nnz = static_cast<int>(out_crows[crow_idx + 1] - out_crows[crow_idx]);
  if (row_nnz == 0) return;

  int kIteration = (row_nnz + warpSize - 1) / warpSize;

  T mul_result = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    mul_result += out_values[row_first + idx] * dout_values[row_first + idx];
  }
  T sum = phi::funcs::warpReduceSum<T>(mul_result, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    dx_values[row_first + idx] =
        (dout_values[row_first + idx] - sum) * out_values[row_first + idx];
  }
}

template <typename T, typename Context>
void SoftmaxCsrGradKernel(const Context& dev_ctx,
                          const SparseCsrTensor& out,
                          const SparseCsrTensor& dout,
                          int axis,
                          SparseCsrTensor* dx) {
  PADDLE_ENFORCE_EQ(axis,
                    -1,
                    phi::errors::Unimplemented(
                        "SparseCsrTensor only support axis=-1 for softmax, "
                        "which is faster when reading data by row (axis=-1)"));
  EmptyLikeCsrKernel<T, Context>(dev_ctx, dout, dx);

  auto out_dim = out.dims();
  auto out_rank = out_dim.size();

  int total_row_number = 1;
  int row_number = 1;
  for (int i = 0; i < out_rank - 1; ++i) {
    total_row_number *= out_dim[i];
    if (i == out_rank - 2) {
      row_number = out_dim[i];
    }
  }

  dim3 grid((total_row_number + 3) / 4);
  dim3 block(32, 4);

  PD_VISIT_BASE_INTEGRAL_TYPES(
      out.crows().dtype(), "SoftmaxCsrGradKernel", ([&] {
        SoftmaxGradGpuKernel<T, data_t><<<grid, block, 0, dev_ctx.stream()>>>(
            out.crows().data<data_t>(),
            out.values().data<T>(),
            dout.values().data<T>(),
            dx->mutable_values()->data<T>(),
            row_number,
            total_row_number);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(softmax_csr_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCsrGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}
