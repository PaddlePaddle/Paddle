#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/softmax_grad_kernel.h"

#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/equal.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/full_kernel.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/sparse/softmax.cu.h"
#include "paddle/phi/kernels/softmax_grad_kernel.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT = int>
__global__ void SoftmaxGradGpuKernel(const IntT* out_crows,
                                     const T* out_values,
                                     const T* dout_values,
                                     T* dx_values,
                                     int row_number,
                                     int total_row_number) {
  // dx = (dout - sum(dout * out)) * out
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  int non_zero_idx = threadIdx.x;
  if (row >= total_row_number) return;
  int cur_batch = row / row_number;
  int crow_idx = cur_batch * (row_number + 1) + (row % row_number);
  int cur_batch_offset = 0;
  for (int i = 1; i < cur_batch + 1; ++i) {
    cur_batch_offset += out_crows[i * (row_number + 1) - 1];
  }
  int row_first = cur_batch_offset + static_cast<int>(out_crows[crow_idx]);
  int row_nnz = static_cast<int>(out_crows[crow_idx + 1] - out_crows[crow_idx]);
  if (row_nnz == 0) return;

  int kIteration = (row_nnz + warpSize - 1) / warpSize;

  T mul_result = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    mul_result += out_values[row_first + idx] * dout_values[row_first + idx];
  }
  T sum = phi::funcs::WarpReduceSum<T>(mul_result, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    dx_values[row_first + idx] =
        (dout_values[row_first + idx] - sum) * out_values[row_first + idx];
  }
}

template <typename T, typename Context>
void SoftmaxCsrGradKernel(const Context& dev_ctx,
                          const SparseCsrTensor& out,
                          const SparseCsrTensor& dout,
                          int axis,
                          SparseCsrTensor* dx) {
  PADDLE_ENFORCE_EQ(axis,
                    -1,
                    phi::errors::Unimplemented(
                        "SparseCsrTensor only support axis=-1 for softmax, "
                        "which is faster when reading data by row (axis=-1)"));
  EmptyLikeCsrKernel<T, Context>(dev_ctx, dout, dx);

  auto out_dim = out.dims();
  auto out_rank = out_dim.size();

  int total_row_number = 1;
  int row_number = 1;
  for (int i = 0; i < out_rank - 1; ++i) {
    total_row_number *= out_dim[i];
    if (i == out_rank - 2) {
      row_number = out_dim[i];
    }
  }

  dim3 grid((total_row_number + 3) / 4);
  dim3 block(32, 4);

  PD_VISIT_BASE_INTEGRAL_TYPES(
      out.crows().dtype(), "SoftmaxCsrGradKernel", ([&] {
        SoftmaxGradGpuKernel<T, data_t><<<grid, block, 0, dev_ctx.stream()>>>(
            out.crows().data<data_t>(),
            out.values().data<T>(),
            dout.values().data<T>(),
            dx->mutable_values()->data<T>(),
            row_number,
            total_row_number);
      }));
}

template <typename T, typename IntT>
__global__ void SoftmaxCooGradGPURawKernel(IntT* sorted_pool_indices,
                                           IntT size,
                                           IntT* pool_sizes,
                                           IntT* pool_offsets,
                                           IntT nvalues,
                                           IntT grad_nnz,
                                           IntT* grad_offsets,
                                           IntT* out_offsets,
                                           IntT* lower_bound_values,
                                           T* values,
                                           T* out_values,
                                           T* grad_values) {
  int tid = threadIdx.x;
  int blkid = blockIdx.x;
  int blksz = blockDim.x;
  int gridsz = gridDim.x;

  int index = tid + blkid * blksz;
  int step = blksz * gridsz;

  while (index < size) {
    IntT offset = pool_offsets[index];
    IntT* pool_indices = sorted_pool_indices + offset;
    IntT pool_indices_size = pool_sizes[index];

    for (IntT k = 0; k < nvalues; k++) {
      T tmp_row{0};

      /* Compute tmp = - sum_j output_j * grad_j */
      for (IntT p = 0; p < pool_indices_size; p++) {
        auto i = pool_indices[p];
        auto cur_out_value = out_values + i * nvalues;
        auto j = lower_bound_values[i];

        /* Update `tmp_row` accumulator only when limits and pools are valid */
        if (j < grad_nnz && (out_offsets[i] == grad_offsets[j])) {
          auto cur_grad_value = grad_values + j * nvalues;
          tmp_row -= (*(cur_out_value + k)) * (*(cur_grad_value + k));
        }
      }

      /* Compute grad_input = output * (grad + tmp)*/
      for (IntT p = 0; p < pool_indices_size; p++) {
        auto i = pool_indices[p];
        auto cur_out_value = out_values + i * nvalues;
        auto cur_value = values + i * nvalues;
        auto j = lower_bound_values[i];
        if (j < grad_nnz && (out_offsets[i] == grad_offsets[j])) {
          auto cur_grad_value = grad_values + j * nvalues;
          cur_value[k] =
              (*(cur_out_value + k)) * (*(cur_grad_value + k) + tmp_row);
        } else {
          cur_value[k] = (*(cur_out_value + k)) * tmp_row;
        }
      }
    }
    index += step;
  }
}

template <typename T, typename IntT, typename Context>
void SoftmaxCooGradGPUKernel(const Context& dev_ctx,
                             const SparseCooTensor& out,
                             const SparseCooTensor& dout,
                             int axis,
                             SparseCooTensor* dx) {
  using thrust_ptr = thrust::device_ptr<IntT>;
  auto out_indices = out.indices();
  auto out_values = out.values();
  auto out_values_ptr = out_values.data<T>();
  const auto output_indices_dims = out.indices().dims();
  const auto out_dims = out.dims();
  auto sparse_dim = out.sparse_dim();
  auto sizes = phi::vectorize<IntT>(out_dims);
  auto grad_indices = dout.indices();
  auto grad_values = dout.values();
  auto grad_values_ptr = grad_values.data<T>();
  auto out_nnz = out.nnz();
  auto grad_nnz = dout.nnz();
  auto place = dev_ctx.GetPlace();
  auto stream = dev_ctx.stream();

  *(dx->mutable_indices()) = out_indices;
  DenseTensor* values = dx->mutable_values();
  values->Resize(out_dims);
  values->set_meta(out_values.meta());
  dev_ctx.template Alloc<T>(values);
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  set_zero(dev_ctx, values, static_cast<T>(0.0f));

  DenseTensor out_offsets = phi::funcs::sparse::GetOffsets<IntT, Context>(
      dev_ctx, out_indices, sizes, static_cast<IntT>(-1));
  auto out_offsets_ptr = out_offsets.data<IntT>();
  DenseTensor grad_offsets = phi::funcs::sparse::GetOffsets<IntT, Context>(
      dev_ctx, grad_indices, sizes, static_cast<IntT>(-1));
  auto grad_offsets_ptr = grad_offsets.data<IntT>();

#ifdef PADDLE_WITH_HIP
  const auto& policy = thrust::hip::par.on(dev_ctx.stream());
  bool is_same_offset = thrust::equal(thrust::hip::par.on(dev_ctx.stream()),
#else
  const auto& policy = thrust::cuda::par.on(dev_ctx.stream());
  bool is_same_offset = thrust::equal(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                                      out_offsets_ptr,
                                      out_offsets_ptr + out_offsets.numel(),
                                      grad_offsets_ptr);

  int dim = axis < 0 ? out_dims.size() + axis : axis;
  if (dim >= sparse_dim) {
    if (is_same_offset) {
      SoftmaxGradKernel<T, Context>(
          dev_ctx, out_values, grad_values, dim - sparse_dim + 1, values);
    } else {
      DenseTensor cur_out_values, cur_grad_values, cur_values;
      cur_out_values.Resize(phi::make_ddim({grad_nnz}));
      dev_ctx.template Alloc<T>(&cur_out_values);
      cur_grad_values.Resize(phi::make_ddim({grad_nnz}));
      dev_ctx.template Alloc<T>(&cur_grad_values);
      cur_values.Resize(phi::make_ddim({grad_nnz}));
      dev_ctx.template Alloc<T>(&cur_values);

      for (IntT i = 0; i < out_nnz; i++) {
        auto low =
            thrust::lower_bound(grad_offsets_ptr,
                                grad_offsets_ptr + grad_offsets.dims()[0],
                                out_offsets_ptr[i]);

        auto j = *low - (*grad_offsets_ptr);
        if (j < grad_nnz && out_offsets_ptr[i] == grad_offsets_ptr[j]) {
          memory_utils::Copy(place,
                             out_values_ptr + i * grad_nnz,
                             place,
                             cur_out_values.data<T>(),
                             grad_nnz * sizeof(T),
                             stream);

          memory_utils::Copy(place,
                             grad_values_ptr + i * grad_nnz,
                             place,
                             cur_grad_values.data<T>(),
                             grad_nnz * sizeof(T),
                             stream);

          SoftmaxGradKernel<T, Context>(dev_ctx,
                                        cur_out_values,
                                        cur_grad_values,
                                        dim - sparse_dim,
                                        &cur_values);

          memory_utils::Copy(place,
                             cur_values.data<T>(),
                             place,
                             values->data<T>() + i * grad_nnz,
                             grad_nnz * sizeof(T),
                             stream);
        }
      }
    }
    return;
  }

  auto nnz = out.nnz();
  IntT nvalues = std::accumulate(sizes.begin() + sparse_dim,
                                 sizes.end(),
                                 static_cast<IntT>(1),
                                 std::multiplies<>());

  DenseTensor values_2(*values);
  values_2.Resize(phi::make_ddim({nnz, nvalues}));

  DenseTensor out_values_2(out_values);
  out_values_2.Resize(phi::make_ddim({nnz, nvalues}));

  DenseTensor grad_values_2(grad_values);
  grad_values_2.Resize(phi::make_ddim({nnz, nvalues}));

  DenseTensor sorted_indices;
  DenseTensor pool_offsets;
  DenseTensor pool_sizes;

  std::tie(sorted_indices, pool_offsets, pool_sizes, std::ignore) =
      phi::funcs::sparse::ComputePoolMax<T, IntT, Context, false>(
          dev_ctx, out_indices, values_2, sizes, nvalues, dim);

  DenseTensor bound =
      phi::Empty<IntT>(dev_ctx, {static_cast<IntT>(out_offsets.dims()[0])});
  IntT* bound_ptr = bound.data<IntT>();
  thrust::lower_bound(policy,
                      thrust_ptr(grad_offsets_ptr),
                      thrust_ptr(grad_offsets_ptr + grad_offsets.dims()[0]),
                      thrust_ptr(out_offsets_ptr),
                      thrust_ptr(out_offsets_ptr) + out_offsets.dims()[0],
                      thrust_ptr(bound.data<IntT>()));

  auto pool_size = pool_offsets.dims()[0];
  int block_size = phi::funcs::sparse::GetNumThreads(pool_size);
  const int grid_size = (pool_size + block_size - 1) / block_size;
  SoftmaxCooGradGPURawKernel<T, IntT>
      <<<grid_size, block_size, 0, stream>>>(sorted_indices.data<IntT>(),
                                             pool_size,
                                             pool_sizes.data<IntT>(),
                                             pool_offsets.data<IntT>(),
                                             nvalues,
                                             grad_nnz,
                                             grad_offsets.data<IntT>(),
                                             out_offsets.data<IntT>(),
                                             bound_ptr,
                                             values_2.data<T>(),
                                             out_values_2.data<T>(),
                                             grad_values_2.data<T>());
}

template <typename T, typename Context>
void SoftmaxCooGradKernel(const Context& dev_ctx,
                          const SparseCooTensor& out,
                          const SparseCooTensor& dout,
                          int axis,
                          SparseCooTensor* dx) {
  PD_VISIT_BASE_INTEGRAL_TYPES(
      out.indices().dtype(), "SoftmaxCooGradGPUKernel", ([&] {
        SoftmaxCooGradGPUKernel<T, data_t, Context>(
            dev_ctx, out, dout, axis, dx);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(softmax_csr_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCsrGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}

PD_REGISTER_KERNEL(softmax_coo_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCooGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
