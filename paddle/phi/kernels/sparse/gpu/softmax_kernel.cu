#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/softmax_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/activation_functor.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT = int>
__global__ void SoftmaxGpuKernel(const IntT* x_crows,
                                 const T* x_values,
                                 T* out_values,
                                 int row_number,
                                 int total_row_number) {
  // out = exp(x-x_max) / sum(exp(x-x_max))
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  int non_zero_idx = threadIdx.x;
  if (row >= total_row_number) return;
  int cur_batch = row / row_number;
  int crow_idx = cur_batch * (row_number + 1) + (row % row_number);
  int cur_batch_offset = 0;
  for (int i = 1; i < cur_batch + 1; ++i) {
    cur_batch_offset += x_crows[i * (row_number + 1) - 1];
  }
  int row_first = cur_batch_offset + static_cast<int>(x_crows[crow_idx]);
  int row_nnz = static_cast<int>(x_crows[crow_idx + 1] - x_crows[crow_idx]);
  if (row_nnz == 0) return;

  int kIteration = (row_nnz + warpSize - 1) / warpSize;

  T max_val = -std::numeric_limits<T>::infinity();
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    T val = x_values[row_first + idx];
    if (val > max_val) {
      max_val = val;
    }
  }
  T row_max_val = phi::funcs::warpReduceMax<T>(max_val, 0xFFFFFFFF);

  T exp_sum = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    auto functor = phi::funcs::CudaExpFunctor<T>();
    T exp = functor(x_values[row_first + idx] - row_max_val);
    exp_sum += exp;
    out_values[row_first + idx] = exp;
  }
  T row_exp_sum = phi::funcs::warpReduceSum<T>(exp_sum, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    out_values[row_first + idx] = out_values[row_first + idx] / row_exp_sum;
  }
}

template <typename T, typename Context>
void SoftmaxCsrKernel(const Context& dev_ctx,
                      const SparseCsrTensor& x,
                      int axis,
                      SparseCsrTensor* out) {
  PADDLE_ENFORCE_EQ(axis,
                    -1,
                    phi::errors::Unimplemented(
                        "SparseCsrTensor only support axis=-1 for softmax, "
                        "which is faster when reading data by row (axis=-1)"));
  EmptyLikeCsrKernel<T, Context>(dev_ctx, x, out);
  auto x_dim = x.dims();
  auto x_rank = x_dim.size();

  int total_row_number = 1;
  int row_number = 1;
  for (int i = 0; i < x_rank - 1; ++i) {
    total_row_number *= x_dim[i];
    if (i == x_rank - 2) {
      row_number = x_dim[i];
    }
  }

  dim3 grid((total_row_number + 3) / 4);
  dim3 block(32, 4);

  PD_VISIT_BASE_INTEGRAL_TYPES(x.crows().dtype(), "CsrSoftmaxKernel", ([&] {
                                 SoftmaxGpuKernel<T, data_t>
                                     <<<grid, block, 0, dev_ctx.stream()>>>(
                                         x.crows().data<data_t>(),
                                         x.values().data<T>(),
                                         out->mutable_values()->data<T>(),
                                         row_number,
                                         total_row_number);
                               }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(softmax_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCsrKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}
