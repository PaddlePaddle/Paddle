#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/softmax_kernel.h"

#include <thrust/device_ptr.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/full_kernel.h"
#include "paddle/phi/kernels/funcs/activation_functor.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/reduce_functor.h"
#include "paddle/phi/kernels/funcs/sparse/softmax.cu.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/softmax_kernel.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT = int>
__global__ void SoftmaxGpuKernel(const IntT* x_crows,
                                 const T* x_values,
                                 T* out_values,
                                 int row_number,
                                 int total_row_number) {
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  int non_zero_idx = threadIdx.x;
  if (row >= total_row_number) return;
  int cur_batch = row / row_number;
  int crow_idx = cur_batch * (row_number + 1) + (row % row_number);
  int cur_batch_offset = 0;
  for (int i = 1; i < cur_batch + 1; ++i) {
    cur_batch_offset += x_crows[i * (row_number + 1) - 1];
  }
  int row_first = cur_batch_offset + static_cast<int>(x_crows[crow_idx]);
  int row_nnz = static_cast<int>(x_crows[crow_idx + 1] - x_crows[crow_idx]);
  if (row_nnz == 0) return;

  int kIteration = (row_nnz + warpSize - 1) / warpSize;

  T max_val = -std::numeric_limits<T>::infinity();
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    T val = x_values[row_first + idx];
    if (val > max_val) {
      max_val = val;
    }
  }
  T row_max_val = phi::funcs::WarpReduceMax<T>(max_val, 0xFFFFFFFF);

  T exp_sum = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    auto functor = phi::funcs::CudaExpFunctor<T>();
    T exp = functor(x_values[row_first + idx] - row_max_val);
    exp_sum += exp;
    out_values[row_first + idx] = exp;
  }
  T row_exp_sum = phi::funcs::WarpReduceSum<T>(exp_sum, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    out_values[row_first + idx] = out_values[row_first + idx] / row_exp_sum;
  }
}

template <typename T, typename Context>
void SoftmaxCsrKernel(const Context& dev_ctx,
                      const SparseCsrTensor& x,
                      int axis,
                      SparseCsrTensor* out) {
  PADDLE_ENFORCE_EQ(axis,
                    -1,
                    phi::errors::Unimplemented(
                        "SparseCsrTensor only support axis=-1 for softmax, "
                        "which is faster when reading data by row (axis=-1)"));
  EmptyLikeCsrKernel<T, Context>(dev_ctx, x, out);
  auto x_dim = x.dims();
  auto x_rank = x_dim.size();

  int total_row_number = 1;
  int row_number = 1;
  for (int i = 0; i < x_rank - 1; ++i) {
    total_row_number *= x_dim[i];
    if (i == x_rank - 2) {
      row_number = x_dim[i];
    }
  }

  dim3 grid((total_row_number + 3) / 4);
  dim3 block(32, 4);

  PD_VISIT_BASE_INTEGRAL_TYPES(x.crows().dtype(), "CsrSoftmaxKernel", ([&] {
                                 SoftmaxGpuKernel<T, data_t>
                                     <<<grid, block, 0, dev_ctx.stream()>>>(
                                         x.crows().data<data_t>(),
                                         x.values().data<T>(),
                                         out->mutable_values()->data<T>(),
                                         row_number,
                                         total_row_number);
                               }));
}

template <typename T, typename IntT>
__global__ void SoftmaxCooGPURawKernel(IntT* sorted_pool_indices,
                                       IntT* pool_sizes,
                                       IntT* pool_offsets,
                                       IntT nvalues,
                                       T* input_values,
                                       T* output_values,
                                       int total_rows) {
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  if (row >= total_rows) return;

  int tid = threadIdx.x;
  int index = row / nvalues;
  int j = row % nvalues;
  IntT offset = pool_offsets[index];
  IntT* pool_indices = sorted_pool_indices + offset;
  IntT pool_indices_size = pool_sizes[index];

  int kIteration = (pool_indices_size + warpSize - 1) / warpSize;
  T max_val = -std::numeric_limits<T>::infinity();
  for (int k = 0; k < kIteration; ++k) {
    int idx = tid + k * warpSize;
    if (idx >= pool_indices_size) break;

    auto i = pool_indices[idx];
    auto cur_value = input_values + j + nvalues * i;
    if (*cur_value > max_val) {
      max_val = *cur_value;
    }
  }
  T row_max_val = phi::funcs::WarpReduceMax<T>(max_val, 0xFFFFFFFF);

  T exp_sum = 0;
  for (int k = 0; k < kIteration; ++k) {
    int idx = tid + k * warpSize;
    if (idx >= pool_indices_size) break;

    auto i = pool_indices[idx];
    auto cur_value = input_values + j + nvalues * i;
    auto cur_out_value = output_values + i * nvalues + j;

    auto functor = phi::funcs::CudaExpFunctor<T>();
    T exp = functor(*cur_value - row_max_val);
    exp_sum += exp;
    *cur_out_value = exp;
  }
  T row_exp_sum = phi::funcs::WarpReduceSum<T>(exp_sum, 0xFFFFFFFF);
  row_exp_sum = 1.0 / row_exp_sum;

  for (int k = 0; k < kIteration; ++k) {
    int idx = tid + k * warpSize;
    if (idx >= pool_indices_size) break;
    auto i = pool_indices[idx];
    auto cur_out_value = output_values + i * nvalues + j;
    *cur_out_value *= row_exp_sum;
  }
}

template <typename T, typename IntT, typename Context>
void SoftmaxCooGPUKernel(const Context& dev_ctx,
                         const SparseCooTensor& x,
                         int axis,
                         SparseCooTensor* out) {
  auto indices = x.indices();
  auto values = x.values();
  const auto x_dims = x.dims();
  const std::vector<IntT> sizes = phi::vectorize<IntT>(x_dims);
  const auto sparse_dim = x.sparse_dim();
  const IntT x_nnz = x.nnz();
  DenseTensor out_indices(indices);
  DenseTensor out_values = EmptyLike<T, Context>(dev_ctx, values);
  out->SetMember(out_indices, out_values, x.dims(), x.coalesced());

  int dim = axis < 0 ? x_dims.size() + axis : axis;

  /* If dim is greater than or equal to sparse_dim, the dense softmax is used.
   */
  if (dim >= sparse_dim) {
    SoftmaxKernel<T, Context>(
        dev_ctx, values, dim - sparse_dim + 1, &out_values);
    return;
  }

  auto stream = dev_ctx.stream();
  IntT nvalues = std::accumulate(sizes.begin() + sparse_dim,
                                 sizes.end(),
                                 static_cast<IntT>(1),
                                 std::multiplies<>());

  auto values_2 = values.Resize({x_nnz, nvalues});
  auto out_values_2 = out_values.Resize({x_nnz, nvalues});

  /* Compute independent pools of indices */
  DenseTensor sorted_indices;
  DenseTensor pool_offsets;
  DenseTensor pool_sizes;
  std::tie(sorted_indices, pool_offsets, pool_sizes, std::ignore) =
      phi::funcs::sparse::ComputePoolMax<T, IntT, Context, false>(
          dev_ctx, indices, values_2, sizes, nvalues, static_cast<IntT>(dim));

  auto pool_size = pool_offsets.dims()[0];
  auto out_values_ptr = out_values.data<T>();
  auto values_ptr = values.data<T>();
  int total_rows = pool_size * nvalues;
  dim3 grid((total_rows + 15) / 16);
  dim3 block(32, 16);
  SoftmaxCooGPURawKernel<T, IntT>
      <<<grid, block, 0, stream>>>(sorted_indices.data<IntT>(),
                                   pool_sizes.data<IntT>(),
                                   pool_offsets.data<IntT>(),
                                   nvalues,
                                   values_ptr,
                                   out_values_ptr,
                                   total_rows);
}

template <typename T, typename Context>
void SoftmaxCooKernel(const Context& dev_ctx,
                      const SparseCooTensor& x,
                      int axis,
                      SparseCooTensor* out) {
  PD_VISIT_BASE_INTEGRAL_TYPES(
      x.indices().dtype(), "SoftmaxCooGPUKernel", ([&] {
        SoftmaxCooGPUKernel<T, data_t, Context>(dev_ctx, x, axis, out);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(softmax_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCsrKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}

PD_REGISTER_KERNEL(softmax_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCooKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
