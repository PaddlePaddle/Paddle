#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/activation_functor.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"
#include "paddle/phi/kernels/sparse/softmax_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename IntT = int>
__global__ void SoftmaxGpuKernel(const IntT* x_crows,
                                 const T* x_values,
                                 T* out_values,
                                 int row_number) {
  // out = exp(x-x_max) / sum(exp(x-x_max))
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  int non_zero_idx = threadIdx.x;
  if (row >= row_number) return;
  int row_first = static_cast<int>(x_crows[row]);
  int row_nnz = static_cast<int>(x_crows[row + 1] - x_crows[row]);
  if (row_nnz == 0) return;

  int kIteration = (row_nnz + warpSize - 1) / warpSize;

  T max_val = -std::numeric_limits<T>::infinity();
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    if (max_val < x_values[row_first + idx]) {
      max_val = x_values[row_first + idx];
    }
  }
  T row_max_val = phi::funcs::warpReduceMax<T>(max_val, 0xFFFFFFFF);

  T exp_sum = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    auto functor = phi::funcs::CudaExpFunctor<T>();
    out_values[row_first + idx] =
        functor(x_values[row_first + idx] - row_max_val);
    exp_sum += functor(x_values[row_first + idx] - row_max_val);
  }
  T row_exp_sum = phi::funcs::warpReduceSum<T>(exp_sum, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = non_zero_idx + i * warpSize;
    if (idx >= row_nnz) break;

    out_values[row_first + idx] = out_values[row_first + idx] / row_exp_sum;
  }
}

template <typename T, typename Context>
void SoftmaxCsrKernel(const Context& dev_ctx,
                      const SparseCsrTensor& x,
                      int axis,
                      SparseCsrTensor* out) {
  PADDLE_ENFORCE_EQ(axis,
                    -1,
                    phi::errors::Unimplemented(
                        "SparseCsrTensor only support axis=-1 for softmax, "
                        "which is faster when reading data by row (axis=-1)"));
  EmptyLikeCsrKernel<T, Context>(dev_ctx, x, out);

  auto x_dim = x.dims();
  int row_number = 1;
  for (int i = 0; i < x_dim.size() - 1; ++i) {
    row_number *= x_dim[i];
  }
  dim3 grid((row_number + 3) / 4);
  dim3 block(32, 4);

  DenseTensor tmp_tensor =
      phi::EmptyLike<T, Context>(dev_ctx, x.non_zero_elements());

  PD_VISIT_INTEGRAL_TYPES(x.non_zero_crows().dtype(), "CsrSoftmaxKernel", ([&] {
                            SoftmaxGpuKernel<T, data_t>
                                <<<grid, block, 0, dev_ctx.stream()>>>(
                                    x.non_zero_crows().data<data_t>(),
                                    x.non_zero_elements().data<T>(),
                                    out->mutable_non_zero_elements()->data<T>(),
                                    row_number);
                          }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(softmax_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SoftmaxCsrKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}
