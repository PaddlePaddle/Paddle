#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/concat_kernel.h"
#include "glog/logging.h"
#include "paddle/phi/backends/cpu/cpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/meta_tensor.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/infermeta/multiary.h"
#include "paddle/phi/kernels/concat_kernel.h"
#include "paddle/phi/kernels/full_kernel.h"
#include "paddle/phi/kernels/funcs/concat_and_split_functor.h"
#include "paddle/phi/kernels/funcs/concat_funcs.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"

namespace phi {
namespace sparse {

template <typename IndexT>
struct DArray {
 public:
  IndexT* d_array{nullptr};

  __device__ inline const void* operator[](int i) const { return d_array[i]; }

  DArray() = default;

  DArray(const phi::GPUContext& ctx, const std::vector<IndexT>& host_array) {
    // copy offsets to device
    auto d_array_tensor = memory_utils::Alloc(
        dev_ctx.GetPlace(),
        sizeof(IndexT) * host_array.size(),
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

    d_array = reinterpret_cast<IndexT*>(d_array_tensor->ptr());

    memory_utils::Copy(dev_ctx.GetPlace(),
                       d_array,
                       phi::CPUPlace(),
                       host_array.data(),
                       sizeof(IndexT) * host_array.size(),
                       dev_ctx.stream());
  }
}

template <typename T>
struct PointerToPointer {
 public:
  void** ins_addr{nullptr};
  __device__ inline const void* operator[](int i) const { return ins_addr[i]; }

  PointerToPointer() = default;
  PointerToPointer(const phi::GPUContext& ctx,
                   size_t in_num const T** pre_alloced_host_ptr) {
    auto* dev_ins_ptr = phi::memory_utils::Alloc(
        ctx.GetPlace(),
        in_num * sizeof(T*),
        phi::Stream(reinterpret_cast<phi::StreamId>(ctx.stream())));
    auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
        pre_alloced_host_ptr, in_num);
    memory_utils::Copy(ctx.GetPlace(),
                       (*dev_ins_ptr)->ptr(),
                       phi::CPUPlace(),
                       restored,
                       in_num * sizeof(T*),
                       ctx.stream());
    ins_addr = reinterpret_cast<void**>((*dev_ins_ptr)->ptr());
  }
};

static void check_cat_sparse_dims(const SparseCooTensor* t,
                                  int64_t pos,
                                  DDim dims,
                                  int64_t axis,
                                  int64_t sparse_dim,
                                  int64_t dense_dim) {
  PADDLE_ENFORCE_EQ(t->sparse_dim(),
                    sparse_dim,
                    "All tensors must have the same sparse_dim ",
                    sparse_dim,
                    ", but tensor at position ",
                    pos,
                    " has ",
                    t->sparse_dim());
  PADDLE_ENFORCE_EQ(t->dense_dim(),
                    dense_dim,
                    "All tensors must have the same dense_dim ",
                    dense_dim,
                    ", but tensor at position ",
                    pos,
                    " has ",
                    t->dense_dim());
}

// ConcatTensorWithDifferentShape<IndexT, MovSize, decltype(ptr_col_array)>
template <typename IndexT>
__global__ void ConcatCooSetIndicesKernel(const IndexT out_nnz,
                                          const IndexT* indice_offset,
                                          const IndexT* col_length,
                                          IndexT* output) {
  IndexT curr_segment = 0;

  // #define CUDA_KERNEL_LOOP_TYPE(i, num, index_type)
  // int64_t __index__ =
  //     static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  // int64_t __stride__ = static_cast<int64_t>(blockDim.x) * gridDim.x;
  // for (index_type i = __index__; __index__ < (num);
  //      __index__ += __stride__, i = __index__)
  CUDA_KERNEL_LOOP_TYPE(tid_x, out_nnz, IndexT) {
    IndexT curr_col_offset = col_length[curr_segment + 1];
    // curr_offset 初始化到最接近tid的对一轮
    // 感觉这里的逻辑是让代码对应到最近tid那一段,毕竟每一轮tid都要递增
    while (curr_col_offset <= tid_x) {
      ++curr_segment;
      curr_col_offset = col_length[curr_segment + 1];
    }
    output[axis * out_nnz + tid_x] += indice_offset[curr_segment];
  }
}

template <typename IndexT, typename PointerWrapperT, typename DarrayWrapperT>
__global__ void ConcatCsr2D0ASetCrowsKernel(const IndexT total_crows_length,
                                            PointerWrapperT in_crows_data,
                                            DarrayWrapperT crows_length,
                                            IndexT* out_crows_data) {
  IndexT curr_segment = 0;

  CUDA_KERNEL_LOOP_TYPE(tid_x, total_crows_length, IndexT) {
    if (tid_x == 0) {
      out_crows_data[0] = 0;
    }
    // 优化
    IndexT curr_col_offset = crows_length[curr_segment + 1];
    IndexT curr_offset = crows_length[curr_segment];
    // curr_offset 初始化到最接近tid的对一轮
    // 感觉这里的逻辑是让代码对应到最近tid那一段,毕竟每一轮tid都要递增
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = crows_length[curr_segment + 1];
    }
    IndexT local_col = tid_x - curr_offset;
    // 注意这里tid_x对应的起始位置不包含crows的第0位,同理,在in_crows中也是如此
    output[tid_x + 1] =
        in_crows_data[curr_segment][local_col + 1] + crows_length[curr_segment];
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsrGet2DRowsNnzKernel(const IndexT total_rows,
                                            const IndexT rows,
                                            PointerWrapperT in_crows_data,
                                            IndexT* in_rows_nnzs) {
  CUDA_KERNEL_LOOP_TYPE(tid_x, total_rows, IndexT) {
    // 优化
    IndexT curr_offset = tid_x / rows;
    IndexT index = tid_x % rows;
    in_rows_nnzs[tid] = in_crows_data[index][curr_offset + 1] -
                        in_crows_data[index][curr_offset];
  }
}

template <typename T,
          typename IndexT,
          typename PointerWrapperT,
          typename PointerWrapperIndexT,
          typename DarrayWrapperT>
__global__ void ConcatCsr2D1ASetValueKernel(const IndexT total_nnz,
                                            const size_t in_num,
                                            PointerWrapperT in_values_data,
                                            PointerWrapperIndexT in_cols_data,
                                            IndexT* in_rows_nnzs_data,
                                            IndexT* in_rows_index,
                                            DarrayWrapperT col_offsets,
                                            T* out_values,
                                            IndexT* out_cols, ) {
  IndexT curr_segment = 0;
  IndexT curr_col_offset = 0;
  IndexT curr_offset = 0;
  CUDA_KERNEL_LOOP_TYPE(tid_x, total_nnz, IndexT) {
    // 优化
    // 这里in_rows_nnzs_data保存的是当前的每行的nnz个数,而不是递增的值
    curr_col_offset = curr_offset + in_rows_nnzs_data[curr_segment + 1];
    // curr_offset 初始化到最接近tid的对一轮
    // 感觉这里的逻辑是让代码对应到最近tid那一段,毕竟每一轮tid都要递增
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset += in_rows_nnzs_data[curr_segment + 1];
    }
    IndexT local_col = tid_x - curr_offset;
    Index total_offset = 0;
    Index index = curr_segment % rows;
    for (Index i = index; i <= curr_segment; i += in_num) {
      total_offset += in_rows_nnzs_data[i];
    }
    total_offset += local_col;
    out_values[tid] = in_values_data[index][total_offset];
    out_cols[tid] = in_cols_data[index][total_offset] + col_offsets[index];
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsr2D1ASetCrowsKernel(const IndexT crows_nums,
                                            const size_t in_num,
                                            PointerWrapperT in_crows_data,
                                            IndexT* out_crows_data) {
  IndexT total_crows = 0;
  // 优化
  CUDA_KERNEL_LOOP_TYPE(tid_x, crows_nums, IndexT) {
    for (int i = 0; i != in_num; i++) {
      total_crows += in_crows_data[i][tid_x];
    }

    output[tid_x] = total_crows;
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsrGet3D1ACrowsKernel(
    const IndexT total_crows,
    const size_t in_num,
    const IndexT batch,
    PointerWrapperT in_crows_data,

    DarrayWrapperT in_rows,  // in_rows表示每一个的列数
    IndexT* in_matrix_nnx) {
  // 获取每轮(batch),也就是一个matrix下的nnz的数目
  // 对应于crows下每一个0开始到rows下的最后一位的数值
  // 例如 crows_data = [0,1,3,5, 8 , 0, 3, 4, 5 ,6] 这里两个需要获取的值是8和6.
  CUDA_KERNEL_LOOP_TYPE(tid_x, total_crows, IndexT) {
    // total_crows == batch* in_num
    IndexT index = tid_x / batch;
    IndexT b = tid_x % batch;
    IndexT pos = b * (in_rows[index] + 1) - 1;
    in_matrix_nnx[tid] = in_crows_data[index][pos];
    // TODO(bapijun) 计算 in_matrix_nnx_offset
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsr3D1ASetCrowsKernel(
    const IndexT out_crows_size,
    const size_t in_num,
    const IndexT batch,
    PointerWrapperT in_crows_data,
    DarrayWrapperT in_matrix_nnx_offset,  // 每个matrix中nnz的个数
    DarrayWrapperT in_rows_offsets,       //
    IndexT* out_crows) {
  // index表示位于第几个输入的
  IndexT index = 0;
  IndexT next_offset = 0;
  IndexT curr_offset = 0;
  // 每一轮batch中的0
  if (b == 0 && local_col == 0) {
    out_crows[in_nnz_offsets[b]] = 0;
  }
  CUDA_KERNEL_LOOP_TYPE(tid_x, out_crows_size, IndexT) {
    // out_crows_size == batch *
    // (各个in_num的rows的和)//注意这里第一位就让第一位处理即可
    IndexT b = tid_x % batch;
    IndexT rows_size = tid_x / batch;
    next_offset = in_rows_offsets[index + 1];
    curr_offset = in_rows_offsets[index];
    // curr_offset 初始化到最接近tid的对一轮
    // 感觉这里的逻辑是让代码对应到最近tid那一段,毕竟每一轮tid都要递增
    while (next_offset <= rows_size) {
      curr_offset = next_offset;
      ++index;
      next_offset = in_rows_offsets[index + 1];
    }
    // IndexT  rows = in_rows_offsets[index];
    IndexT local_col = rows_size - curr_offset;
    // 在 in_matrix_nnx_offset中位置p =  index * batch + b
    // out_crows中由于存在index个0
    out_crows[tid + index] = in_crows_data[index][local_col] +
                             in_matrix_nnx_offset[index * batch + b];
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsr3D1ASetValuesColsKernel(
    const IndexT total_nnz,
    const size_t in_num,
    const IndexT batch,
    PointerWrapperT in_values_data,
    DarrayWrapperT
        in_matrix_nnx_offset,  // 每个matrix(也就是每轮batch)中nnz的累加值
    DarrayWrapperT in_batch_offsets,  // 这里是
    IndexT* out_values) {
  IndexT b = 0;
  IndexT next_offset = 0;
  IndexT curr_offset = 0;
  // total_nnz = in_matrix_nnx的和 =
  CUDA_KERNEL_LOOP_TYPE(tid_x, total_nnz, IndexT) {
    // in_batch_crows 表示每一轮
    next_offset += in_batch_crows[b + 1];
    curr_offset += in_batch_crows[b];
    // curr_offset 初始化到最接近tid的对一轮
    // 感觉这里的逻辑是让代码对应到最近tid那一段,毕竟每一轮tid都要递增
    while (next_offset <= tid_x) {
      curr_offset = next_offset;
      ++b;
      next_offset += in_batch_crows[b + 1];
    }
    IndexT local_col = tid_x - curr_offset;
    // p =  index * batch + b
    IndexT j = (b - 1) * in_num;

    next_offset = in_matrix_nnx_offset[j + 1];
    curr_offset = in_matrix_nnx_offset[j];

    while (next_offset <= local_col) {
      curr_offset = next_offset;
      ++j;
      next_offset = in_matrix_nnx_offset[j + 1];
    }
    IndexT index = j - (b - 1) * in_num;
    IndexT local_col2 = local_col - curr_offset;
    // local_col2 += index_offset[b];//每一轮k的叠加值
    for (int k = b; k > 0; k--) {
      local_col2 += in_crows_data[index][k * in_rows[index]];
    }

    out_values[tid_x] = in_values[index][local_col2];
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsr3D2ASetCrowsKernel(
    const IndexT crows_num,
    const size_t in_num,
    PointerWrapperT in_crows_data,
    DarrayWrapperT
        in_matrix_nnx_offset,  // 每个matrix(也就是每轮batch)中nnz的累加值

    DarrayWrapperT in_batch_offsets,  // 这里是
    IndexT* out_crows) {
  // total_nnz = in_matrix_nnx的和 =
  CUDA_KERNEL_LOOP_TYPE(tid_x, crows_num, IndexT) {
    // TODO(bapijun) 使用什么方式优化
    IndexT total = 0;
    for (int i = 0; i < in_num; i++) {
      total = in_crows_data[i][tid_x];
    }
    out_crows[tid_x] = total;
  }
}

template <typename IndexT, typename PointerWrapperT>
__global__ void ConcatCsr3D2ASetvaluesKernel(
    const IndexT max_nnz,
    const size_t in_num,
    const IndexT batch,
    PointerWrapperT in_crows_data,
    DarrayWrapperT in_matrix_nnx_offset,  // 每个matrix(也就是每轮batch)中nnz的累加值
    PointerWrapperT in_max_batch_nnz,
    DarrayWrapperT in_batch_offsets,  // 这里是每一轮batch的最大值
    IndexT* out_values) {
  // total_nnz = in_matrix_nnx的和 =

  // 根据i和b获取的最大nnz 可以在上一个核函数获取
  IndexT b = 0;
  IndexT next_offset = 0;
  IndexT curr_offset = 0;
  IndexT now_nnz = in_max_batch_nnz[tid_z][tid_y];
  CUDA_KERNEL_LOOP_TYPE(tid_x, now_nnz, IndexT) {
    // TODO(bapijun) 使用什么方式优化
    next_offset += in_batch_crows[b + 1];
    curr_offset += in_batch_crows[b];
    // curr_offset 初始化到最接近tid的对一轮
    // 感觉这里的逻辑是让代码对应到最近tid那一段,毕竟每一轮tid都要递增
    while (next_offset <= tid_x) {
      curr_offset = next_offset;
      ++b;
      next_offset += in_batch_crows[b + 1];
    }
    IndexT local_col = tid_x - curr_offset;

    IndexT j = (b - 1) * in_num;
    next_offset = in_matrix_nnx_offset[j + in_num];
    curr_offset = in_matrix_nnx_offset[j];

    while (next_offset <= local_col) {
      curr_offset = next_offset;
      j = j+in_num;
      next_offset = in_matrix_nnx_offset[j + in_num];
    }
    
    // 根据in_batch_offsets 获取到当前的batch下的最大值
  }
}

template <typename T, typename IntT, typename Context>
void ConcatCooGPUKernel(const Context& dev_ctx,
                        const std::vector<const SparseCooTensor*>& x,
                        const Scalar& axis_scalar,
                        SparseCooTensor* out) {
  std::vector<DenseTensor> indices;
  std::vector<DenseTensor> values;
  std::vector<phi::DDim> x_dims;
  IntT in_num = ins.size();
  IntT axis = axis_scalar.to<IntT>();
  axis = phi::funcs::ComputeAxis(axis, x[0]->dims().size());
  IntT sparse_dim = x[0]->sparse_dim();
  IntT dense_dim = x[0]->dense_dim();

  DDim dims = x[0]->dims();
  DenseTensor out_indices;
  DenseTensor out_values;
  // 替换成 使用指针的形式
  funcs::ConcatFunctor<Context, T> concat_functor_value;
  funcs::ConcatFunctor<Context, IntT> concat_functor_indice;
  IntT pos = 0;
  for (const auto* t : x) {
    check_cat_sparse_dims(t, pos, dims, axis, sparse_dim, dense_dim);
    x_dims.push_back(t->dims());
    pos++;
  }
  // 迁移到对应的代码里面去,或者查看其他方式
  EmptyLikeCooKernel<T, Context>(dev_ctx, *x[0], out);
  phi::DDim out_dims = phi::funcs::ComputeAndCheckShape(true, x_dims, axis);
  if (axis < sparse_dim) {
    int64_t out_nnz = 0, out_cols = 0;
    std::vector<IndexT> indice_offsets(in_num, 0);
    std::vector<IndexT> in_cols(in_num + 1, 0);
    in_nnz[0] = 0;
    IntT i = 0;
    for (const auto* t : x) {
      indices.emplace_back(t->indices());
      values.emplace_back(t->values());
      out_nnz += t->nnz();
      in_cols[i + 1] = out_nnz;
      out_cols += t.dims()[axis];
      indice_offsets[i] = out_cols;
    }
    out_indices = phi::Empty<IndexT, Context>(dev_ctx, {sparse_dim, out_nnz});

    DDim v_dim = x[0]->values().dims();
    v_dim[0] = out_nnz;
    IntArray v_shape(v_dim.GetMutable(), v_dim.size());
    out_values = phi::Empty<T, Context>(dev_ctx, v_shape);

    // copy indice_offsets to device
    auto d_indice_offsets_tensor = memory_utils::Alloc(
        dev_ctx.GetPlace(),
        sizeof(IndexT) * indice_offsets.size(),
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

    IndexT* d_indice_offsets =
        reinterpret_cast<IndexT*>(d_indice_offsets_tensor->ptr());

    memory_utils::Copy(dev_ctx.GetPlace(),
                       d_indice_offsets,
                       phi::CPUPlace(),
                       indice_offsets.data(),
                       sizeof(int64_t) * indice_offsets.size(),
                       dev_ctx.stream());

    // copy in_cols to device
    auto d_in_cols_tensor = memory_utils::Alloc(
        dev_ctx.GetPlace(),
        sizeof(IndexT) * in_cols.size(),
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

    IndexT* d_in_cols = reinterpret_cast<IndexT*>(d_in_cols_tensor->ptr());

    memory_utils::Copy(dev_ctx.GetPlace(),
                       d_in_cols,
                       phi::CPUPlace(),
                       in_cols.data(),
                       sizeof(int64_t) * in_cols.size(),
                       dev_ctx.stream());

    // 因为在前面进行了检查,所以这个维度的nnz都一样
    concat_functor_indice(dev_ctx, indices, static_cast<int>(1), &out_indices);

    concat_functor_value(dev_ctx, values, static_cast<int>(0), &out_values);

    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, out_nnz, 1);
    ConcatCooSetIndicesKernel<IntT><<<config.block_per_grid.x,
                                      config.thread_per_block.x,
                                      0,
                                      dev_ctx.stream()>>>(
        out_nnz,
        const IndexT* d_indice_offsets const IndexT* d_in_cols IndexT* output);

    out->SetMember(out_crows, out_cols, out_values, out_dims);
  } else {
    int64_t values_dim = axis - sparse_dim + 1;

    int64_t total_size = 0;
    for (auto& r : x) {
      total_size += r->values().dims()[values_dim];
    }
    DDim zeros_sizes = x[0]->values().dims();
    int64_t cumulative_size = 0;

    for (const auto* t : x) {
      zeros_sizes[0] = t->values().dims()[0];
      zeros_sizes[values_dim] = cumulative_size;
      cumulative_size += t->values().dims()[values_dim];
      // z1 z2是全0的向量
      DenseTensor z1 =
          phi::Full<T, Context>(dev_ctx, common::vectorize(zeros_sizes), 0);
      zeros_sizes[values_dim] = total_size - cumulative_size;
      DenseTensor z2 =
          phi::Full<T, Context>(dev_ctx, common::vectorize(zeros_sizes), 0);
      std::vector<DenseTensor> now_values;
      now_values.push_back(z1);
      now_values.push_back(t->values());
      now_values.push_back(z2);
      auto concat_value =
          std::make_shared<DenseTensor>();  // 创建DenseTensor的智能指针
      concat_functor_value(dev_ctx, now_values, values_dim, concat_value.get());
      // 用 phi::funcs::StridedNumelCopyWithAxis<T, Context>
      values.push_back(*concat_value);
      indices.push_back(t->indices());
    }
    concat_functor_indice(dev_ctx, indices, static_cast<int>(1), &out_indices);
    concat_functor_value(dev_ctx, values, static_cast<int>(0), &out_values);

    out->SetMember(out_indices, out_values, out_dims, x[0]->coalesced());
  }
}

template <typename T, typename Context>
void ConcatCooKernel(const Context& dev_ctx,
                     const std::vector<const SparseCooTensor*>& x,
                     const Scalar& axis_scalar,
                     SparseCooTensor* out) {
  int64_t out_nnz = 0;
  for (const auto* t : x) {
    out_nnz += t.nnz();
  }
  if (out_nnz < std::numeric_limits<int32_t>::max()) {
    ConcatCooGPUKernel<T, int32_t>(dev_ctx, x, axis_scalar, out);
  } else {
    ConcatCooGPUKernel<T, int64_t>(dev_ctx, x, axis_scalar, out);
  }
}

template <typename T, typename IntT, typename Context>
void ConcatCsrGPU2D0A(const Context& dev_ctx,
                      const std::vector<const SparseCooTensor*>& x,
                      size_t in_num,
                      const phi::DDim& out_dims,
                      T* out_values_data,
                      IntT* out_cols_data,
                      const DenseTensor& out_values,
                      const DenseTensor& out_cols,
                      const std::vector<const T*>& values_data_vec,
                      const std::vector<const int64_t*>& cols_data_vec,
                      const std::vector<const int64_t*>& crows_data_vec,
                      SparseCooTensor* out) {
  // 到这里为止所有的代码可以合并到前文去,这里留在这里只是为了未来方便调试

  // 除了第一个0 之外,按照row的次数叠加
  int64_t out_crows_size = 0;
  std::vector<int64_t> nnz_vec(in_num, 0);
  std::vector<int64_t> crows_length(in_num + 1, 0);
  for (size_t i = 0; i < in_num; i++) {
    nnz_vec.push_back(t->nnz());
    int64_t crows_num = static_cast<int64_t>(x[i]->dims()[0]) + 1;
    out_crows_size += crows_num;
    crows_length[i + 1] = out_crows_size;
  }
  out_crows_size++;
  DenseTensor out_crows = phi::Empty<int64_t>(dev_ctx, {out_crows_size});
  int64_t* out_crows_data = out_crows.data<int64_t>();

  auto gpu_place = dev_ctx.GetPlace();
  auto stream = dev_ctx.stream();
  int64_t value_offset = 0;
  // 用合并的concat方法替代
  for (size_t i = 0; i < in_num; i++) {
    int nnz = nnz_vec[i];
    // nnz == 0 的特殊情况,此时out_values_data指针很可能是错误的
    memory_utils::Copy(gpu_place,
                       out_values_data + value_offset,
                       gpu_place,
                       values_data_vec[i],
                       nnz * sizeof(T),
                       stream);
    memory_utils::Copy(gpu_place,
                       out_cols_data + value_offset,
                       gpu_place,
                       cols_data_vec[i],
                       nnz * sizeof(int64_t),
                       stream);
    value_offset += nnz;
  }
  const int64_t** crows_data_vec_data = crows_data_vec.data();

  PointerToPointer<int64_t> crows_ptr_array(
      dev_ctx, in_num, crows_data_vec_data);
  DArray<int64_t> d_crows_length(dev_ctx, crows_length);

  // 这里由于每一轮需要的只处理crow中0之后的部分 out_crows_size -1
  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, out_crows_size - 1, 1);

  ConcatCsr2D0ASetCrowsKernel<int64_t,
                              decltype(crows_ptr_array),
                              decltype(d_crows_length)>
      <<<config.block_per_grid.x,
         config.thread_per_block.x,
         0,
         dev_ctx.stream()>>>(
          out_crows_size - 1, crows_ptr_array, d_crows_length, out_crows_data);

  out->SetMember(out_crows, out_cols, out_values, out_dims);
}

void ConcatCsrGPU2D1A<T, IntT, Context>(
    const Context& dev_ctx,
    const std::vector<const SparseCooTensor*>& x,
    size_t in_num,
    const phi::DDim& out_dims,
    SparseCooTensor* out) {
  // 除了第一个0 之外,按照row的次数叠加
  std::vector<int64_t> col_offsets;

  int col_offset = 0 =, total_nnz = 0;
  col_offsets.push_back(col_offset);
  for (const auto* t : x) {
    col_offset += static_cast<int64_t>(x[i]->dims()[1]);
    col_offsets.push_back(col_offset);
    total_nnz += x[i]->nnz();
  }
  IntT rows = static_cast<size_t>(x[0]->dims()[0]);
  out_crows_size = rows + 1;
  int64_t total_rows = rows * in_num;

  DenseTensor out_crows = phi::Empty<IntT>(dev_ctx, {out_crows_size});
  IntT* out_crows_data = out_crows.data<IntT>();

  // 在设备中global memory的保存各个行的nnz个数
  DenseTensor in_rows_nnzs_tensor = phi::Empty<IntT>(dev_ctx, {total_rows});
  IntT* d_in_rows_nnzs_data = in_rows_nnzs_tensor.data<IntT>();

  PointerToPointer<IntT> crows_ptr_array(dev_ctx, in_num, crows_data_vec_data);

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, total_rows, 1);
  ConcatCsrGetRowsNnzKernel<IntT, decltype(crows_ptr_array)>
      <<<config.block_per_grid.x,
         config.thread_per_block.x,
         0,
         dev_ctx.stream()>>>(
          total_rows, rows, crows_ptr_array, d_in_rows_nnzs_data);

  PointerToPointer<T> values_ptr_array(dev_ctx, in_num, values_data_vec);
  PointerToPointer<IntT> cols_ptr_array(dev_ctx, in_num, cols_data_vec);
  DArray<IntT> d_col_offsets(dev_ctx, col_offsets);

  config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, total_nnz, 1);
  ConcatCsr2D1ASetValueKernel<T,
                              IntT,
                              decltype(values_ptr_array),
                              decltype(cols_ptr_array),
                              decltype(d_col_offsets)>
      <<<config.block_per_grid.x,
         config.thread_per_block.x,
         0,
         dev_ctx.stream()>>>(total_nnz,
                             in_num,
                             in_values_data,
                             in_cols_data,
                             d_in_rows_nnzs_data,
                             col_offsets,
                             out_values_data,
                             out_cols_data);

  config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, out_crows_size, 1);

  ConcatCsr2D1ASetCrowsKernel<IntT, decltype(crows_ptr_array)>
      <<<config.block_per_grid.x,
         config.thread_per_block.x,
         0,
         dev_ctx.stream()>>>(
          out_crows_size, in_num, crows_ptr_array, out_crows_data);

  out->SetMember(out_crows, out_cols, out_values, out_dims);
}

template <typename T, typename IntT, typename Context>
void ConcatCsrGPU3D0A(const Context& dev_ctx,
                      const std::vector<const SparseCooTensor*>& x,
                      size_t in_num,
                      const phi::DDim& out_dims,
                      T* out_values_data,
                      IntT* out_cols_data,
                      const DenseTensor& out_values,
                      const DenseTensor& out_cols,
                      const std::vector<const T*>& values_data_vec,
                      const std::vector<const int64_t*>& cols_data_vec,
                      const std::vector<const int64_t*>& crows_data_vec,
                      SparseCooTensor* out) {
  // 到这里为止所有的代码可以合并到前文去,这里留在这里只是为了未来方便调试

  std::vector<DenseTensor> crows;
  std::vector<DenseTensor> values;
  std::vector<DenseTensor> cols;

  for (size_t i = 0; i < in_num; i++) {
    crows.emplace_back(x[i]->crows());
    values.emplace_back(x[i]->values());
    cols.emplace_back(x[i]->cols());
    out_crows_size += x[i]->crows().numel();
  }

  // axis==0 简单拼接所有的三个即可即可完成
  funcs::ConcatFunctor<Context, T> concat_functor;
  concat_functor(dev_ctx, values, static_cast<T>(0), &out_values);
  // cols的形状与value一致
  funcs::ConcatFunctor<Context, int64_t> concat_functor_indices;
  concat_functor_indices(dev_ctx, cols, static_cast<int64_t>(0), &out_cols);
  DenseTensor out_crows = phi::Empty<int64_t>(dev_ctx, {out_crows_size});
  concat_functor_indices(dev_ctx, crows, static_cast<int64_t>(0), &out_crows);

  out->SetMember(out_crows, out_cols, out_values, out_dims);
}

void ConcatCsrGPU3D1A<T, IntT, Context>(
    const Context& dev_ctx,
    const std::vector<const SparseCooTensor*>& x,
    size_t in_num,
    const phi::DDim& out_dims,
    T* out_values_data,
    IntT* out_cols_data,
    const DenseTensor& out_values,
    const DenseTensor& out_cols,
    const std::vector<const T*>& values_data_vec,
    const std::vector<const int64_t*>& cols_data_vec,
    const std::vector<const int64_t*>& crows_data_vec,
    SparseCooTensor* out) {
  // 除了第一个0 之外,按照row的次数叠加
  size_t batch = static_cast<int>(x[0]->dims()[0]);

  out_crows_size = batch;
  for (size_t i = 0; i < in_num; i++) {
    int64_t rows = static_cast<int64_t>(x[i]->dims()[1]);
    crows_numel.push_back(rows + 1);
    out_crows_size += batch * rows;
  }
  DenseTensor out_crows = phi::Empty<int64_t>(dev_ctx, {out_crows_size});
  int64_t* out_crows_data = out_crows.data<int64_t>();

  PointerToPointer<IntT> crows_ptr_array(dev_ctx, in_num, crows_data_vec_data);
  if (batch * in_num < 10) {  // 需要更精确的逻辑
    for (size_t b = 0; b < batch; b++) {
      // 针对每一轮batch的初始化
      out_crows_data[crow_index] = 0;
      crow_index++;
      cumulative_offset = 0;

      for (size_t i = 0; i < in_num; i++) {
        const int64_t* x_crows_ptr = x[i]->crows().data<int64_t>();
        // crows_numel[i] == 第i组的row+1
        int64_t x_crows_nnz = x_crows_ptr[(b + 1) * (crows_numel[i]) - 1];
        now_value_ptr = values_data_vec[i] + values_index[i];
        now_cols_ptr = cols_data_vec[i] + values_index[i];
        values_index[i] += x_crows_nnz;

        if (x_crows_nnz) {
          // nnz == 0 的特殊情况,此时out_values_data指针很可能是错误的
          memory_utils::Copy(cpu_place,
                             out_values_data + value_offset,
                             cpu_place,
                             now_value_ptr,
                             x_crows_nnz * sizeof(T));
          memory_utils::Copy(cpu_place,
                             out_cols_data + value_offset,
                             cpu_place,
                             now_cols_ptr,
                             x_crows_nnz * sizeof(int64_t));
        }

        value_offset += x_crows_nnz;
      }
    }
  } else {
  }

  out->SetMember(out_crows, out_cols, out_values, out_dims);
}

template <typename T, typename Context>
void ConcatCsrGPUKernel(const Context& dev_ctx,
                        const std::vector<const SparseCooTensor*>& x,
                        const Scalar& axis_scalar,
                        SparseCooTensor* out) {
  size_t in_num = x.size();

  int64_t axis = axis_scalar.to<int64_t>();
  axis = phi::funcs::ComputeAxis(axis, x[0]->dims().size());
  // 删掉
  std::vector<phi::DDim> x_dims;
  x_dims.reserve(in_num);
  std::vector<int64_t> crows_numel;
  std::vector<const int64_t*> crows_data_vec;
  std::vector<const T*> values_data_vec;
  std::vector<const int64_t*> cols_data_vec;
  crows_numel.reserve(in_num);
  crows_data_vec.reserve(in_num);
  values_data_vec.reserve(in_num);
  cols_data_vec.reserve(in_num);

  int64_t out_values_size = 0;
  int64_t out_crows_size = 0;
  for (const auto* t : x) {
    // TODO(bapijun) 考虑到nnz = 0的情况,进行补全`

    x_dims.emplace_back(t->dims());
    values_data_vec.push_back(t->values().data<T>());
    cols_data_vec.push_back(t->cols().data<int64_t>());
    // nnz == 0 时候,如果crow = [0] 这样的情况,补全0,避免之后的拼接遗漏
    crows_data_vec.push_back(t->crows().data<int64_t>());
    out_values_size += t->nnz();
  }
  DenseTensor out_values = phi::Empty<T>(dev_ctx, {out_values_size});
  T* out_values_data = out_values.data<T>();
  DenseTensor out_cols = phi::Empty<int64_t>(dev_ctx, {out_values_size});
  int64_t* out_cols_data = out_cols.data<int64_t>();

  phi::DDim out_dims = phi::funcs::ComputeAndCheckShape(true, x_dims, axis);
  int x_dim = x_dims[0].size();
  if (x_dim == 2) {
    if (axis == 0) {
      ConcatCsrGPU2D0A<T, IntT, Context>(dev_ctx,
                                         x,
                                         out_dims,
                                         in_num,
                                         out_values_data,
                                         out_cols_data,
                                         crows_data_vec,
                                         out);
    } else {
      ConcatCsrGPU2D1A<T, IntT, Context>(dev_ctx,
                                         x,
                                         out_dims,
                                         in_num,
                                         out_values_data,
                                         out_cols_data,
                                         crows_data_vec,
                                         out);
    }

  } else if (x_dims.size() == 3) {
    // ConcatCsrGPU3D<T, IntT, Context>(
    //   dev_ctx, x,  axis_scalar, out);
  } else {
    // throw exception
    phi::errors::InvalidArgument(
        "Concat for Sparse CSR Tensor only support 2-D or 3-D, but got %d-D.",
        x_dims.size());
  }
}

template <typename T, typename Context>
void ConcatCsrKernel(const Context& dev_ctx,
                     const std::vector<const SparseCsrTensor*>& x,
                     const Scalar& axis_scalar,
                     SparseCsrTensor* out) {
  int64_t out_nnz = 0;
  for (const auto* t : x) {
    out_nnz += t.nnz();
  }
  if (out_nnz < std::numeric_limits<int32_t>::max()) {
    ConcatCsrGPUKernel<T, int32_t>(dev_ctx, x, axis_scalar, out);
  } else {
    ConcatCsrGPUKernel<T, int64_t>(dev_ctx, x, axis_scalar, out);
  }
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(concat_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::ConcatCooKernel,
                   float,
                   double,
                   bool,
                   int64_t,
                   int,
                   uint8_t,
                   int8_t,
                   int16_t) {}

PD_REGISTER_KERNEL(concat_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::ConcatCsrKernel,
                   float,
                   double,
                   bool,
                   int64_t,
                   int,
                   uint8_t,
                   int8_t,
                   int16_t) {}
