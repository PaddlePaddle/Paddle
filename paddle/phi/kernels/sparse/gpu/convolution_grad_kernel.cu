#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/convolution_grad_kernel.h"

#include "glog/logging.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/copy_kernel.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/scatter.cu.h"
#include "paddle/phi/kernels/funcs/sparse/scatter.cu.h"
#include "paddle/phi/kernels/sparse/gpu/convolution.cu.h"

namespace phi {
namespace sparse {

// rulebook[3, rulebook_len]:
//[
//  [kernel_index],
//  [in_i],
//  [out_i],
//]
// x_grad = out_grad * transpose(kenrel)
// kernel_grad = transpose(x) * out_grad
template <typename T, typename IntT>
void Conv3dGradGPUKernel(const GPUContext& dev_ctx,
                         const SparseCooTensor& x,
                         const DenseTensor& kernel,
                         const SparseCooTensor& out,
                         const SparseCooTensor& out_grad,
                         const std::vector<int>& paddings,
                         const std::vector<int>& dilations,
                         const std::vector<int>& strides,
                         const int groups,
                         const bool subm,
                         const std::string& key,
                         SparseCooTensor* x_grad,
                         DenseTensor* kernel_grad) {
  const auto& kernel_dims = kernel.dims();
  const int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];

  const auto* table = out.table(key);
  const DenseTensor& rulebook = table->first;
  const IntT* rulebook_ptr = rulebook.data<IntT>();

  const int rulebook_len = rulebook.dims()[1];

  DenseTensorMeta in_features_meta(
      x.dtype(), {rulebook_len, in_channels}, DataLayout::NCHW);
  DenseTensorMeta d_x_features_meta(
      x.dtype(), {rulebook_len, in_channels}, DataLayout::NCHW);
  DenseTensorMeta out_grad_features_meta(
      x.dtype(), {rulebook_len, out_channels}, DataLayout::NCHW);
  phi::DenseTensor in_features =
      phi::Empty(dev_ctx, std::move(in_features_meta));
  phi::DenseTensor d_x_features =
      phi::Empty(dev_ctx, std::move(d_x_features_meta));
  phi::DenseTensor out_grad_features =
      phi::Empty(dev_ctx, std::move(out_grad_features_meta));

  T* in_features_ptr = in_features.data<T>();
  T* d_x_features_ptr = d_x_features.data<T>();
  T* out_grad_features_ptr = out_grad_features.data<T>();
  *kernel_grad = phi::EmptyLike<T>(dev_ctx, kernel);
  T* d_kernel_ptr = kernel_grad->data<T>();
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  // set_zero(dev_ctx, kernel_grad, static_cast<T>(0.0f));
  phi::backends::gpu::GpuMemsetAsync(
      d_kernel_ptr, 0, sizeof(T) * kernel_grad->numel(), dev_ctx.stream());

  int half_kernel_size = kernel_size / 2;
  auto blas = phi::funcs::GetBlas<GPUContext, T>(dev_ctx);
  DenseTensor x_grad_indices =
      phi::EmptyLike<IntT>(dev_ctx, x.non_zero_indices());
  DenseTensor x_grad_values = phi::EmptyLike<T>(dev_ctx, x.non_zero_elements());
  T* x_grad_values_ptr = x_grad_values.data<T>();
  // set_zero(dev_ctx, &x_grad_values, static_cast<T>(0.0f));
  phi::backends::gpu::GpuMemsetAsync(x_grad_values_ptr,
                                     0,
                                     sizeof(T) * x_grad_values.numel(),
                                     dev_ctx.stream());
  // set_zero(dev_ctx, &d_x_features, static_cast<T>(0.0f));
  phi::backends::gpu::GpuMemsetAsync(
      d_x_features_ptr, 0, sizeof(T) * d_x_features.numel(), dev_ctx.stream());
  phi::Copy<GPUContext>(dev_ctx,
                        x.non_zero_indices(),
                        dev_ctx.GetPlace(),
                        false,
                        &x_grad_indices);
  x_grad->SetMember(x_grad_indices, x_grad_values, x.dims(), true);

  std::vector<int> offsets(kernel_size + 1);
  const auto& counter = table->second;

  int offset = 0, max_count = 0;
  for (int i = 0; i < kernel_size; i++) {
    offsets[i] = offset;
    offset += counter[i];
    if (i < half_kernel_size) {
      max_count = std::max(max_count, counter[i]);
    }
  }
  offsets[kernel_size] = offset;

  if (subm) {
    phi::funcs::sparse::SubmPreProcess<T, GPUContext>(
        dev_ctx,
        x,
        kernel,
        out_grad.non_zero_elements(),
        in_channels,
        out_channels,
        half_kernel_size,
        kernel_grad,
        &x_grad_values);
    if (max_count == 0) {
      return;
    }
  }

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, rulebook_len, 1);
  DenseTensor unique_value =
      phi::Empty<int>(dev_ctx, {static_cast<int>(x_grad->nnz() * kernel_size)});
  DenseTensor out_index =
      phi::Empty<int>(dev_ctx, {static_cast<int>(rulebook_len)});
  int* out_index_ptr = out_index.data<int>();
  int* unique_value_ptr = unique_value.data<int>();
  hipMemsetAsync(
      out_index_ptr, 0, sizeof(int) * rulebook_len, dev_ctx.stream());

  UpdateOutIndex<<<config.block_per_grid,
                   config.thread_per_block,
                   0,
                   dev_ctx.stream()>>>(
      rulebook_len, kernel_size, rulebook_ptr, out_index_ptr, unique_value_ptr);

  const int VecSize = VecBytes / sizeof(T);
  if (in_channels % VecSize == 0) {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, rulebook_len * in_channels / VecSize, 1);
    GatherKernelV2<T, IntT, VecSize>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                               // rulebook_ptr,
                               out_index_ptr,
                               unique_value_ptr,
                               rulebook_len,
                               kernel_size,
                               in_features_ptr,
                               in_channels);
  } else {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, rulebook_len * in_channels, 1);
    GatherKernelV2<T, IntT, 1>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                               out_index_ptr,
                               unique_value_ptr,
                               rulebook_len,
                               kernel_size,
                               in_features_ptr,
                               in_channels);
  }

  if (out_channels % VecSize == 0) {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, rulebook_len * out_channels / VecSize, 1);
    GatherKernel<T, IntT, VecSize>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(out_grad.non_zero_elements().data<T>(),
                               rulebook_ptr + rulebook_len,
                               out_grad_features_ptr,
                               rulebook_len,
                               out_channels);
  } else {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, rulebook_len * out_channels, 1);
    GatherKernel<T, IntT, 1>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(out_grad.non_zero_elements().data<T>(),
                               rulebook_ptr + rulebook_len,
                               out_grad_features_ptr,
                               rulebook_len,
                               out_channels);
  }

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (counter[i] <= 0 || (subm && i == half_kernel_size)) {
      continue;
    }

    const int M = counter[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + offsets[i] * in_channels;
    T* tmp_out_grad_ptr = out_grad_features_ptr + offsets[i] * out_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * in_channels * out_channels;
    T* tmp_d_x_ptr = d_x_features_ptr + offsets[i] * in_channels;
    T* tmp_d_kernel_ptr = d_kernel_ptr + i * in_channels * out_channels;

    // call gemm: d_kernel = transpose(x) * out_grad
    // (in_channels, n) * (n, out_channels)
    blas.GEMM(CblasTrans,
              CblasNoTrans,
              K,
              N,
              M,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_out_grad_ptr,
              static_cast<T>(0),
              tmp_d_kernel_ptr);

    // call gemm: d_x = out_grad * transpose(kernel)
    // (n, out_channels) * (out_channels, in_channels)
    blas.GEMM(CblasNoTrans,
              CblasTrans,
              M,
              K,
              N,
              static_cast<T>(1),
              tmp_out_grad_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_d_x_ptr);
  }

  // 4. scatter
  //  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
  //      dev_ctx, rulebook_len * in_channels, 1);
  //
  //  phi::funcs::ScatterCUDAKernel<<<config.block_per_grid,
  //                                  config.thread_per_block,
  //                                  0,
  //                                  dev_ctx.stream()>>>(d_x_features_ptr,
  //                                                      rulebook_ptr,
  //                                                      x_grad_values_ptr,
  //                                                      rulebook_len,
  //                                                      in_channels,
  //                                                      false);

  if (in_channels % VecSize == 0) {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, x_grad->nnz() * in_channels / VecSize, 1);
    phi::funcs::sparse::ScatterKernelV2<T, VecSize>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(d_x_features_ptr,
                               out_index.data<int>(),
                               unique_value.data<int>(),
                               x_grad->nnz(),
                               kernel_size,
                               in_channels,
                               x_grad_values_ptr);
  } else {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, x_grad->nnz() * in_channels, 1);
    phi::funcs::sparse::ScatterKernelV2<T, 1>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(d_x_features_ptr,
                               out_index.data<int>(),
                               unique_value.data<int>(),
                               x_grad->nnz(),
                               kernel_size,
                               in_channels,
                               x_grad_values_ptr);
  }
}

template <typename T, typename Context>
void Conv3dGradKernel(const Context& dev_ctx,
                      const SparseCooTensor& x,
                      const DenseTensor& kernel,
                      const SparseCooTensor& out,
                      const SparseCooTensor& out_grad,
                      const std::vector<int>& paddings,
                      const std::vector<int>& dilations,
                      const std::vector<int>& strides,
                      const int groups,
                      const bool subm,
                      const std::string& key,
                      SparseCooTensor* x_grad,
                      DenseTensor* kernel_grad) {
  PD_VISIT_INTEGRAL_TYPES(
      x.non_zero_indices().dtype(), "Conv3dGradGPUKernel", ([&] {
        Conv3dGradGPUKernel<T, data_t>(dev_ctx,
                                       x,
                                       kernel,
                                       out,
                                       out_grad,
                                       paddings,
                                       dilations,
                                       strides,
                                       groups,
                                       subm,
                                       key,
                                       x_grad,
                                       kernel_grad);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(sparse_conv3d_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
