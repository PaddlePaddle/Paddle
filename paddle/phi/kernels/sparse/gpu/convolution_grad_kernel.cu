/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/sparse/convolution_grad_kernel.h"
#include "paddle/phi/kernels/sparse/gpu/convolution.cu.h"

namespace phi {
namespace sparse {

// rulebook[3, rulebook_len]:
//[
//  [kernel_index],
//  [in_i],
//  [out_i],
//]
// x_grad = out_grad * transpose(kenrel)
// kernel_grad = transpose(x) * out_grad
template <typename T, typename Context>
void Conv3dGradKernel(const Context& dev_ctx,
                      const SparseCooTensor& x,
                      const DenseTensor& rulebook,
                      const DenseTensor& kernel,
                      const SparseCooTensor& out_grad,
                      const std::vector<int>& paddings,
                      const std::vector<int>& dilations,
                      const std::vector<int>& strides,
                      const int groups,
                      DenseTensor* x_grad,
                      DenseTensor* kernel_grad) {
  const auto& kernel_dims = kernel.dims();
  const int kernel_size = kernel_dims[0] * kernel_dims[1] * kernel_dims[2];
  const int in_channels = kernel_dims[3];
  const int out_channels = kernel_dims[4];
  const int* rulebook_ptr = rulebook.data<int>();

  const int rulebook_len = rulebook.dims()[1];

  DenseTensorMeta in_features_meta(
      x.dtype(), {rulebook_len, in_channels}, DataLayout::NCHW);
  DenseTensorMeta d_x_features_meta(
      x.dtype(), {rulebook_len, in_channels}, DataLayout::NCHW);
  DenseTensorMeta out_grad_features_meta(
      x.dtype(), {rulebook_len, out_channels}, DataLayout::NCHW);
  phi::DenseTensor in_features =
      phi::Empty(dev_ctx, std::move(in_features_meta));
  phi::DenseTensor d_x_features =
      phi::Empty(dev_ctx, std::move(d_x_features_meta));
  phi::DenseTensor out_grad_features =
      phi::Empty(dev_ctx, std::move(out_grad_features_meta));

  dev_ctx.Alloc(
      &in_features, in_features.dtype(), sizeof(T) * in_features.numel());
  T* in_features_ptr = in_features.data<T>();
  dev_ctx.Alloc(
      &d_x_features, d_x_features.dtype(), sizeof(T) * d_x_features.numel());
  T* d_x_features_ptr = d_x_features.data<T>();
  dev_ctx.Alloc(&out_grad_features,
                out_grad_features.dtype(),
                sizeof(T) * out_grad_features.numel());
  T* out_grad_features_ptr = out_grad_features.data<T>();
  kernel_grad->Resize(kernel_dims);
  dev_ctx.Alloc(
      kernel_grad, kernel_grad->dtype(), kernel_grad->numel() * sizeof(T));
  T* d_kernel_ptr = kernel_grad->data<T>();
  phi::funcs::SetConstant<Context, int> set_zero;
  set_zero(dev_ctx, kernel_grad, 0);

  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, rulebook_len * in_channels, 1);
  GatherKernel<T, int><<<config.block_per_grid.x,
                         config.thread_per_block.x,
                         0,
                         dev_ctx.stream()>>>(x.non_zero_elements().data<T>(),
                                             rulebook_ptr + rulebook_len,
                                             in_features_ptr,
                                             rulebook_len,
                                             in_channels);

  config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, rulebook_len * out_channels, 1);
  GatherKernel<T, int><<<config.block_per_grid.x,
                         config.thread_per_block.x,
                         0,
                         dev_ctx.stream()>>>(
      out_grad.non_zero_elements().data<T>(),
      rulebook_ptr + rulebook_len * 2,
      out_grad_features_ptr,
      rulebook_len,
      out_channels);

  auto blas = phi::funcs::GetBlas<Context, T>(dev_ctx);
  std::vector<int> offsets(kernel_size + 1), counter(kernel_size, 0),
      h_counter(rulebook_len, 0);
  phi::backends::gpu::GpuMemcpyAsync(&h_counter[0],
                                     rulebook_ptr,
                                     rulebook_len * sizeof(int),
#ifdef PADDLE_WITH_HIP
                                     hipMemcpyDeviceToHost,
#else
                                     hipMemcpyDeviceToHost,
#endif

                                     dev_ctx.stream());
  dev_ctx.Wait();

  for (int i = 0; i < rulebook_len; i++) {
    counter[h_counter[i]] += 1;
  }
  int offset = 0;
  for (int i = 0; i < kernel_size; i++) {
    offsets[i] = offset;
    offset += counter[i];
  }
  offsets[kernel_size] = offset;

  const T* kernel_ptr = kernel.data<T>();
  for (int i = 0; i < kernel_size; i++) {
    if (counter[i] <= 0) {
      continue;
    }

    const int M = counter[i];
    const int K = in_channels;
    const int N = out_channels;
    T* tmp_in_ptr = in_features_ptr + offsets[i] * in_channels;
    T* tmp_out_grad_ptr = out_grad_features_ptr + offsets[i] * out_channels;
    const T* tmp_kernel_ptr = kernel_ptr + i * in_channels * out_channels;
    T* tmp_d_x_ptr = d_x_features_ptr + offsets[i] * out_channels;
    T* tmp_d_kernel_ptr = d_kernel_ptr + i * in_channels * out_channels;

    // call gemm: d_kernel = transpose(x) * out_grad
    // (in_channels, n) * (n, out_channels)
    blas.GEMM(CblasTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1),
              tmp_in_ptr,
              tmp_out_grad_ptr,
              static_cast<T>(0),
              tmp_d_kernel_ptr);

    // call gemm: d_x = out_grad * transpose(kernel)
    // (n, out_channels) * (out_channels, in_channels)
    blas.GEMM(CblasNoTrans,
              CblasTrans,
              M,
              K,
              N,
              static_cast<T>(1),
              tmp_out_grad_ptr,
              tmp_kernel_ptr,
              static_cast<T>(0),
              tmp_d_x_ptr);
  }

  // 4. scatter
  x_grad->Resize(x.non_zero_elements().dims());
  dev_ctx.Alloc(x_grad, x_grad->dtype(), sizeof(T) * x_grad->numel());
  T* x_grad_values_ptr = x_grad->data<T>();

  DenseTensor out_index = phi::Empty<int, Context>(dev_ctx);
  DenseTensor unique_key = phi::Empty<int, Context>(dev_ctx);
  DenseTensor unique_value = phi::Empty<int, Context>(dev_ctx);
  unique_key.ResizeAndAllocate({rulebook_len});
  out_index.ResizeAndAllocate({rulebook_len});
  unique_value.ResizeAndAllocate({rulebook_len});
  dev_ctx.Alloc(
      &unique_key, unique_key.dtype(), sizeof(int) * unique_key.numel());
  dev_ctx.Alloc(&out_index, out_index.dtype(), sizeof(int) * out_index.numel());
  dev_ctx.Alloc(
      &unique_value, unique_value.dtype(), sizeof(int) * unique_value.numel());

  SortedAndUniqueIndex(dev_ctx,
                       rulebook_ptr + rulebook_len,
                       rulebook_len,
                       &out_index,
                       &unique_key,
                       &unique_value);

  config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, rulebook_len * in_channels, 1);

  ScatterKernel<T><<<config.block_per_grid.x,
                     config.thread_per_block.x,
                     0,
                     dev_ctx.stream()>>>(d_x_features_ptr,
                                         unique_value.data<int>(),
                                         out_index.data<int>(),
                                         x.nnz(),
                                         rulebook_len,
                                         in_channels,
                                         x_grad_values_ptr);
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(sparse_conv3d_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::Conv3dGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
  kernel->InputAt(3).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
