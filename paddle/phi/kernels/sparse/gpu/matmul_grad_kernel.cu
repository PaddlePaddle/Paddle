/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/matmul_grad_kernel.h"

#include <vector>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/sparse/sparse_blas.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"
#include "paddle/phi/kernels/sparse/sparse_utils_kernel.h"
#include "paddle/phi/kernels/transpose_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename Context>
void MatmulCooDenseGradKernel(const Context& dev_ctx,
                              const SparseCooTensor& x,
                              const DenseTensor& y,
                              const DenseTensor& dout,
                              SparseCooTensor* dx,
                              DenseTensor* dy) {
#if CUDA_VERSION >= 11030
  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);

  // dx{SparseCoo} = dout{Dense} * y'{Dense}
  if (dx) {
    // 'hipsparseSDDMM' only support CSR now, so use COO->CSR->COO,
    // which will increase some expenses.
    EmptyLikeCooKernel<T, Context>(dev_ctx, x, dx);
    SparseCsrTensor dx_csr = CooToCsr<T, Context>(dev_ctx, *dx);
    sparse_blas.SDDMM(
        false, true, static_cast<T>(1), dout, y, static_cast<T>(0), &dx_csr);
    CsrToCooKernel<T, Context>(dev_ctx, dx_csr, dx);
  }

  // dy{Dense} = x'{SparseCoo} * dout{Dense}
  if (dy) {
    MetaTensor meta_dy(dy);
    meta_dy.set_dims(y.dims());
    meta_dy.set_dtype(y.dtype());
    dev_ctx.template Alloc<T>(dy);

    sparse_blas.SPMM(
        true, false, static_cast<T>(1), x, dout, static_cast<T>(0), dy);
  }
#else
  PADDLE_THROW(phi::errors::Unimplemented(
      "backward of 'sparse.matmul' use hipsparseSDDMM, which is supported from "
      "CUDA 11.3"));
#endif
}

template <typename T, typename Context>
void MatmulCsrDenseGradKernel(const Context& dev_ctx,
                              const SparseCsrTensor& x,
                              const DenseTensor& y,
                              const DenseTensor& dout,
                              SparseCsrTensor* dx,
                              DenseTensor* dy) {
#if CUDA_VERSION >= 11030
  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);

  // dx{SparseCsr} = dout{Dense} * y'{Dense}
  if (dx) {
    // InferMeta of SparseCsrTensor 'dx', CreateLikeInferMeta
    EmptyLikeCsrKernel<T, Context>(dev_ctx, x, dx);

    sparse_blas.SDDMM(
        false, true, static_cast<T>(1), dout, y, static_cast<T>(0), dx);
  }

  // dy{Dense} = x'{SparseCsr} * dout{Dense}
  if (dy) {
    // InferMeta of DenseTensor 'dy'
    MetaTensor meta_dy(dy);
    meta_dy.set_dims(y.dims());
    meta_dy.set_dtype(y.dtype());

    dev_ctx.template Alloc<T>(dy);

    sparse_blas.SPMM(
        true, false, static_cast<T>(1), x, dout, static_cast<T>(0), dy);
  }
#else
  PADDLE_THROW(phi::errors::Unimplemented(
      "backward of 'sparse.matmul' use hipsparseSDDMM, which is supported from "
      "CUDA 11.3"));
#endif
}

template <typename T, typename Context>
void MaskedMatmulCsrGradKernel(const Context& dev_ctx,
                               const DenseTensor& x,
                               const DenseTensor& y,
                               const SparseCsrTensor& dout,
                               DenseTensor* dx,
                               DenseTensor* dy) {
#if CUDA_VERSION >= 11000
  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);

  // dx{Dense} = dout{SparseCsr} * y'{Dense}
  if (dx) {
    // InferMeta of DenseTensor 'dx'
    MetaTensor meta_dx(dx);
    meta_dx.set_dims(x.dims());
    meta_dx.set_dtype(x.dtype());

    dev_ctx.template Alloc<T>(dx);
    sparse_blas.SPMM(
        false, true, static_cast<T>(1), dout, y, static_cast<T>(0), dx);
  }

  // dy{Dense} = x'{Dense} * dout{SparseCsr}
  // That is: dy'{Dense} = dout'{SparseCsr} * x{Dense}
  if (dy) {
    std::vector<int> trans_dim_vec = phi::vectorize<int>(y.dims());
    size_t rank = trans_dim_vec.size();
    std::swap(trans_dim_vec[rank - 1], trans_dim_vec[rank - 2]);
    DenseTensor trans_dy = phi::Empty<T, Context>(dev_ctx, trans_dim_vec);

    sparse_blas.SPMM(
        true, false, static_cast<T>(1), dout, x, static_cast<T>(0), &trans_dy);

    // InferMeta of DenseTensor 'dy'
    MetaTensor meta_dy(dy);
    meta_dy.set_dims(y.dims());
    meta_dy.set_dtype(y.dtype());

    dev_ctx.template Alloc<T>(dy);

    size_t y_ndim = y.dims().size();
    std::vector<int> axis(y_ndim);
    for (size_t i = 0; i < y_ndim; ++i) {
      axis[i] = i;
    }
    std::swap(axis[y_ndim - 1], axis[y_ndim - 2]);
    TransposeKernel<T, Context>(dev_ctx, trans_dy, axis, dy);
  }
#endif
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(matmul_coo_dense_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::MatmulCooDenseGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}

PD_REGISTER_KERNEL(matmul_csr_dense_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::MatmulCsrDenseGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}

PD_REGISTER_KERNEL(masked_matmul_csr_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::MaskedMatmulCsrGradKernel,
                   float,
                   double) {}
