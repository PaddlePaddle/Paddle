#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/fused_attention_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/activation_functor.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/funcs/sparse/sparse_blas.h"
#include "paddle/phi/kernels/sparse/empty_kernel.h"
#include "paddle/phi/kernels/sparse/matmul_kernel.h"
#include "paddle/phi/kernels/sparse/sparse_utils_kernel.h"

namespace phi {
namespace sparse {

#define PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, size, HINT, ...) \
  case size: {                                                 \
    constexpr int HINT = size;                                 \
    __VA_ARGS__();                                             \
    break;                                                     \
  }

#define VISIT_ATTN_SFOTMAX(SIZE, NAME, ...)                                 \
  [&] {                                                                     \
    const auto& __size__ = SIZE;                                            \
    switch (__size__) {                                                     \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 1, KBufferSize, __VA_ARGS__)    \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 2, KBufferSize, __VA_ARGS__)    \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 3, KBufferSize, __VA_ARGS__)    \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 4, KBufferSize, __VA_ARGS__)    \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 8, KBufferSize, __VA_ARGS__)    \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 12, KBufferSize, __VA_ARGS__)   \
      PRIVATE_CASE_VISIT_ATTN_SOFTMAX(NAME, 16, KBufferSize, __VA_ARGS__)   \
      default:                                                              \
        PD_THROW("function " #NAME " is not implemented for columns>512 "); \
    }                                                                       \
  }()

template <typename T, int BufferSize>
__global__ void AttnSoftmaxGpuKernel(const int64_t* x_crows,
                                     const int64_t* x_cols,
                                     const T* x_values,
                                     const T* kp_mask,
                                     const T* attn_mask,
                                     T* out_values,
                                     int M,
                                     int total_row_num,
                                     float scale,
                                     int num_heads,
                                     int batch_nnz) {
  // out = exp(x-x_max) / sum(exp(x-x_max))
  int row = blockIdx.x * blockDim.y + threadIdx.y;
  if (row >= total_row_num) return;

  int cur_batch = row / M;
  int cur_row = row % M;
  int crow_idx = cur_batch * (M + 1) + cur_row;
  int row_first = cur_batch * batch_nnz + static_cast<int>(x_crows[crow_idx]);
  int row_nnz = static_cast<int>(x_crows[crow_idx + 1] - x_crows[crow_idx]);
  if (row_nnz == 0) return;

  T buffer[BufferSize] = {0};
  int kIteration = (row_nnz + WARP_SIZE - 1) / WARP_SIZE;

  T max_val = -std::numeric_limits<T>::infinity();
  for (int i = 0; i < kIteration; ++i) {
    bool mask = false;
    int idx = threadIdx.x + i * WARP_SIZE;
    if (idx >= row_nnz) break;

    int col_idx = static_cast<int>(x_cols[row_first + idx]);

    if (kp_mask != nullptr &&
        kp_mask[(cur_batch / num_heads) * M + col_idx] == 0) {
      mask = true;
    }
    if (attn_mask != nullptr && attn_mask[cur_row * M + col_idx] == 0) {
      mask = true;
    }

    if (!mask) {
      buffer[i] = x_values[row_first + idx] / scale;
      if (buffer[i] > max_val) {
        max_val = buffer[i];
      }
    }
  }
  T row_max_val = phi::funcs::warpReduceMax<T>(max_val, 0xFFFFFFFF);

  auto functor = phi::funcs::CudaExpFunctor<T>();
  T exp_sum = 0;
  for (int i = 0; i < kIteration; ++i) {
    int idx = threadIdx.x + i * WARP_SIZE;
    if (idx >= row_nnz) break;

    if (buffer[i]) {
      T exp = functor(buffer[i] - row_max_val);
      exp_sum += exp;
      buffer[i] = exp;
    }
  }
  T row_exp_sum = phi::funcs::warpReduceSum<T>(exp_sum, 0xFFFFFFFF);

  for (int i = 0; i < kIteration; ++i) {
    int idx = threadIdx.x + i * WARP_SIZE;
    if (idx >= row_nnz) break;

    if (buffer[i]) {
      out_values[row_first + idx] = buffer[i] / row_exp_sum;
    } else {
      out_values[row_first + idx] = static_cast<T>(0);
    }
  }
}

template <typename T, typename Context>
void FusedAttentionCsrKernel(
    const Context& dev_ctx,
    const DenseTensor& query,
    const DenseTensor& key,
    const DenseTensor& value,
    const SparseCsrTensor& sparse_mask,
    const paddle::optional<DenseTensor>& key_padding_mask,
    const paddle::optional<DenseTensor>& attn_mask,
    DenseTensor* out,
    SparseCsrTensor* softmax) {
#if CUDA_VERSION >= 11070
  /* Check Shape */
  auto q_dim = query.dims();
  auto q_rank = q_dim.size();

  int total_row_num = 1;
  int batch_num = 1;
  for (int i = 0; i < q_rank - 1; ++i) {
    total_row_num *= q_dim[i];
    if (i < q_rank - 2) {
      batch_num *= q_dim[i];
    }
  }
  int M = q_dim[q_rank - 2];
  int N = q_dim[q_rank - 1];

  PADDLE_ENFORCE_EQ(query.dims().size(),
                    4,
                    phi::errors::InvalidArgument(" 'query' must be 4D Tensor"));
  PADDLE_ENFORCE_EQ(key.dims().size(),
                    4,
                    phi::errors::InvalidArgument(" 'key' must be 4D Tensor"));
  PADDLE_ENFORCE_EQ(value.dims().size(),
                    4,
                    phi::errors::InvalidArgument(" 'value' must be 4D Tensor"));

  PADDLE_ENFORCE_EQ(
      sparse_mask.dims().size(),
      3,
      phi::errors::InvalidArgument("dense shape of 'sparse_mask' must be "
                                   "[batch_size*num_heads, seq_len, seq_len]"));
  PADDLE_ENFORCE_EQ(
      sparse_mask.dims()[0],
      q_dim[0] * q_dim[1],
      phi::errors::InvalidArgument("dense shape of 'sparse_mask' must be "
                                   "[batch_size*num_heads, seq_len, seq_len]"));
  PADDLE_ENFORCE_EQ(
      sparse_mask.dims()[1],
      M,
      phi::errors::InvalidArgument("dense shape of 'sparse_mask' must be "
                                   "[batch_size*num_heads, seq_len, seq_len]"));
  PADDLE_ENFORCE_EQ(
      sparse_mask.dims()[2],
      M,
      phi::errors::InvalidArgument("dense shape of 'sparse_mask' must be "
                                   "[batch_size*num_heads, seq_len, seq_len]"));

  const auto kp_mask_ptr = key_padding_mask.get_ptr();
  if (kp_mask_ptr) {
    PADDLE_ENFORCE_EQ(
        kp_mask_ptr->dims().size(),
        2,
        phi::errors::InvalidArgument(
            "shape of 'key_padding_mask' must be [batch_size, seq_len]"));
    PADDLE_ENFORCE_EQ(
        kp_mask_ptr->dims()[0],
        q_dim[0],
        phi::errors::InvalidArgument(
            "shape of 'key_padding_mask' must be [batch_size, seq_len]"));
    PADDLE_ENFORCE_EQ(
        kp_mask_ptr->dims()[1],
        M,
        phi::errors::InvalidArgument(
            "shape of 'key_padding_mask' must be [batch_size, seq_len]"));
  }

  const auto attn_mask_ptr = attn_mask.get_ptr();
  if (attn_mask_ptr) {
    PADDLE_ENFORCE_EQ(attn_mask_ptr->dims().size(),
                      2,
                      phi::errors::InvalidArgument(
                          "shape of 'attn_mask' must be [seq_len, seq_len]"));
    PADDLE_ENFORCE_EQ(attn_mask_ptr->dims()[0],
                      M,
                      phi::errors::InvalidArgument(
                          "shape of 'attn_mask' must be [seq_len, seq_len]"));
    PADDLE_ENFORCE_EQ(attn_mask_ptr->dims()[1],
                      M,
                      phi::errors::InvalidArgument(
                          "shape of 'attn_mask' must be [seq_len, seq_len]"));
  }

  /* Step1: SDD Matmul, reuse */
  SparseCsrTensor sdd_result;
  EmptyLikeCsrKernel<T, Context>(dev_ctx, sparse_mask, &sdd_result);
  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);
  sparse_blas.SDDMM(false,
                    true,
                    static_cast<T>(1),
                    query,
                    key,
                    static_cast<T>(0),
                    &sdd_result);

  /* Step2: Softmax with kp_mask/attn_mask, manualy not reuse */
  EmptyLikeCsrKernel<T, Context>(dev_ctx, sdd_result, softmax);

  int buffer_size;
  if (M < 128) {
    buffer_size = (M + 32 - 1) / 32;
  } else {
    buffer_size = ((M + 128 - 1) / 128) * 4;
  }

  dim3 grid((total_row_num + 3) / 4);
  dim3 block(WARP_SIZE, 4);

  int batch_nnz = sdd_result.nnz() / batch_num;

  VISIT_ATTN_SFOTMAX(buffer_size, "AttnSoftmaxGpuKernel", [&] {
    AttnSoftmaxGpuKernel<T, KBufferSize><<<grid, block, 0, dev_ctx.stream()>>>(
        sdd_result.non_zero_crows().data<int64_t>(),
        sdd_result.non_zero_cols().data<int64_t>(),
        sdd_result.non_zero_elements().data<T>(),
        kp_mask_ptr ? kp_mask_ptr->data<T>() : nullptr,
        attn_mask_ptr ? attn_mask_ptr->data<T>() : nullptr,
        softmax->mutable_non_zero_elements()->data<T>(),
        M,
        total_row_num,
        std::sqrt(N),
        q_dim[1],
        batch_nnz);
  });

  /* Step3: DSD Matmul, reuse */
  softmax->set_dims(phi::make_ddim({q_dim[0], q_dim[1], q_dim[2], q_dim[2]}));
  MatmulCsrDenseKernel<T, Context>(dev_ctx, *softmax, value, out);
#else
  PADDLE_THROW(
      phi::errors::Unimplemented("forward of 'sparse.nn.functional.attention' "
                                 "use 'hipsparseCsrSetStridedBatch', which is "
                                 "completed supported from CUDA 11.7"));
#endif
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(fused_attention_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::FusedAttentionCsrKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}
