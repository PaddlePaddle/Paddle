/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/copy_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/sparse/sparse_blas.h"
#include "paddle/phi/kernels/sparse/sparse_mm_grad_kernel.h"
#include "paddle/phi/kernels/transpose_kernel.h"

namespace phi {
namespace sparse {

template <typename T, typename Context>
void CsrDenseMatmulGradKernel(const Context& dev_ctx,
                              const SparseCsrTensor& x,
                              const DenseTensor& y,
                              const DenseTensor& dout,
                              SparseCsrTensor* dx,
                              DenseTensor* dy) {
#if CUDA_VERSION >= 11030
  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);

  // dx{SparseCsr} = dout{Dense} * y'{Dense}
  if (dx) {
    // InferMeta of SparseCsrTensor 'dx'
    dx->set_dims(x.dims());

    phi::Copy(dev_ctx,
              x.non_zero_crows(),
              dev_ctx.GetPlace(),
              false,
              dx->mutable_non_zero_crows());
    phi::Copy(dev_ctx,
              x.non_zero_cols(),
              dev_ctx.GetPlace(),
              false,
              dx->mutable_non_zero_cols());

    DenseTensor* values = dx->mutable_non_zero_elements();
    values->Resize(x.non_zero_elements().dims());
    dev_ctx.template Alloc<T>(values);

    sparse_blas.SDDMM(
        false, true, static_cast<T>(1), dout, y, static_cast<T>(0), dx);
  }

  // dy{Dense} = x'{SparseCsr} * dout{Dense}
  if (dy) {
    // InferMeta of DenseTensor 'dy'
    MetaTensor meta_dy(dy);
    meta_dy.set_dims(y.dims());
    meta_dy.set_dtype(y.dtype());

    dev_ctx.template Alloc<T>(dy);

    sparse_blas.DSDMM(
        true, false, static_cast<T>(1), x, dout, static_cast<T>(0), dy);
  }
#else
  PADDLE_THROW(phi::errors::Unimplemented(
      " backward of 'sparse.mm' use hipsparseSDDMM, Only "
      "support it from CUDA 11.3"));
#endif
}

template <typename T, typename Context>
void CsrMaskedMatmulGradKernel(const Context& dev_ctx,
                               const DenseTensor& x,
                               const DenseTensor& y,
                               const SparseCsrTensor& dout,
                               DenseTensor* dx,
                               DenseTensor* dy) {
  auto sparse_blas = phi::funcs::sparse::GetSparseBlas<Context, T>(dev_ctx);

  // dx{Dense} = dout{SparseCsr} * y'{Dense}
  if (dx) {
    // InferMeta of DenseTensor 'dx'
    MetaTensor meta_dx(dx);
    meta_dx.set_dims(x.dims());
    meta_dx.set_dtype(x.dtype());

    dev_ctx.template Alloc<T>(dx);
    sparse_blas.DSDMM(
        false, true, static_cast<T>(1), dout, y, static_cast<T>(0), dx);
  }

  // dy{Dense} = x'{Dense} * dout{SparseCsr}
  // That is: dy'{Dense} = dout'{SparseCsr} * x{Dense}
  if (dy) {
    std::vector<int> trans_dim_vec = phi::vectorize<int>(y.dims());
    size_t rank = trans_dim_vec.size();
    std::swap(trans_dim_vec[rank - 1], trans_dim_vec[rank - 2]);
    DenseTensor trans_dy = phi::Empty<T, Context>(dev_ctx, trans_dim_vec);

    sparse_blas.DSDMM(
        true, false, static_cast<T>(1), dout, x, static_cast<T>(0), &trans_dy);

    // InferMeta of DenseTensor 'dy'
    MetaTensor meta_dy(dy);
    meta_dy.set_dims(y.dims());
    meta_dy.set_dtype(y.dtype());

    dev_ctx.template Alloc<T>(dy);

    size_t y_ndim = y.dims().size();
    std::vector<int> axis(y_ndim);
    for (size_t i = 0; i < y_ndim; ++i) {
      axis[i] = i;
    }
    std::swap(axis[y_ndim - 1], axis[y_ndim - 2]);
    TransposeKernel<T, Context>(dev_ctx, trans_dy, axis, dy);
  }
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(csr_dense_matmul_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CsrDenseMatmulGradKernel,
                   float,
                   double) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}

PD_REGISTER_KERNEL(csr_masked_mm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CsrMaskedMatmulGradKernel,
                   float,
                   double) {}
