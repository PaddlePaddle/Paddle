#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/funcs/sparse/convolution.h"
#include "paddle/phi/kernels/sparse/gpu/convolution.cu.h"
#include "paddle/phi/kernels/sparse/sparse_pool_kernel.h"

namespace phi {
namespace sparse {

template <typename T>
__global__ void MaxPoolCudaKernel(const T* in_features_ptr,
                                  const int* rulebook_ptr,
                                  const int n,
                                  const int rulebook_len,
                                  const int channels,
                                  T* out_features_ptr) {
  CUDA_KERNEL_LOOP_TYPE(i, n * channels, int64_t) {
    int real_i = i / channels;
    int channel_i = i - real_i * channels;
    int in_i = rulebook_ptr[real_i];
    int out_i = rulebook_ptr[real_i + rulebook_len];
    if (out_features_ptr[out_i * channels + channel_i] <
        in_features_ptr[in_i * channels + channel_i]) {
      out_features_ptr[out_i * channels + channel_i] =
          in_features_ptr[in_i * channels + channel_i];
    }
  }
}

/**
 * x: (N, D, H, W, C)
 * kernel: (D, H, W, C, OC)
 * out: (N, D, H, W, OC)
**/
template <typename T, typename Context>
void MaxPoolKernel(const Context& dev_ctx,
                   const SparseCooTensor& x,
                   const std::vector<int>& kernel_sizes,
                   const std::vector<int>& paddings,
                   const std::vector<int>& dilations,
                   const std::vector<int>& strides,
                   SparseCooTensor* out,
                   DenseTensor* rulebook) {
  const auto& x_dims = x.dims();
  int kernel_size = kernel_sizes[0] * kernel_sizes[1] * kernel_sizes[2];
  DDim out_dims = {1, 1, 1, 1, 1};
  phi::funcs::sparse::GetOutShape(
      x_dims, kernel_sizes, paddings, dilations, strides, &out_dims);
  const int in_channels = kernel_sizes[3];

  std::vector<int> offsets(kernel_size + 1), counter(kernel_size);
  DenseTensorMeta counter_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensorMeta offsets_meta(
      DataType::INT32, {kernel_size}, DataLayout::NCHW);
  DenseTensor counter_per_kernel = phi::Empty(dev_ctx, std::move(counter_meta));
  DenseTensor offsets_per_kernel = phi::Empty(dev_ctx, std::move(offsets_meta));
  DenseTensorMeta index_meta(DataType::INT32, {1}, DataLayout::NCHW);
  DenseTensor out_index = phi::Empty(dev_ctx, std::move(index_meta));
  DenseTensor unique_key = phi::Empty(dev_ctx, std::move(index_meta));
  DenseTensor unique_value = phi::Empty(dev_ctx, std::move(index_meta));

  int rulebook_len = ProductRuleBook<T, Context>(dev_ctx,
                                                 x,
                                                 kernel_sizes,
                                                 paddings,
                                                 dilations,
                                                 strides,
                                                 out_dims,
                                                 false,
                                                 rulebook,
                                                 &counter_per_kernel,
                                                 &offsets_per_kernel,
                                                 &out_index,
                                                 &unique_key,
                                                 &unique_value,
                                                 out,
                                                 &counter,
                                                 &offsets);

  const int* rulebook_ptr = rulebook->data<int>();

  T* out_features_ptr = out->mutable_non_zero_elements()->data<T>();
  const T* in_features_ptr = x.non_zero_elements().data<T>();
  // 1. get the min elements of in_features
  const T* result =
      thrust::min_element(thrust::device,
                          in_features_ptr,
                          in_features_ptr + x.non_zero_elements().numel());
  // 2. init the out_features with min elements
  T h_result;
  hipMemcpy(&h_result, result, sizeof(T), hipMemcpyDeviceToHost);
  thrust::fill(thrust::device,
               out_features_ptr,
               out_features_ptr + out->non_zero_elements().numel(),
               h_result);
  for (int i = 0; i < kernel_size; i++) {
    if (counter[i] <= 0) {
      continue;
    }

    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
        dev_ctx, counter[i] * in_channels, 1);
    MaxPoolCudaKernel<T><<<config.block_per_grid.x,
                           config.thread_per_block.x,
                           0,
                           dev_ctx.stream()>>>(
        in_features_ptr,
        rulebook_ptr + offsets[i] + rulebook_len,
        counter[i],
        rulebook_len,
        in_channels,
        out_features_ptr);
  }
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(sparse_maxpool,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::MaxPoolKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}
