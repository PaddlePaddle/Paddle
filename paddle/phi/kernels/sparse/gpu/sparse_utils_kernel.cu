#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/remove.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/sparse/sparse_utils_kernel.h"

namespace phi {
namespace sparse {

template <typename T>
inline __device__ bool DevIsZero(const T* data, const int64_t cols) {
  const T zero = static_cast<T>(0);
  // TODO(zhangkaihuo): check the data is zero or not in parallen when cols > 1
  for (int64_t i = 0; i < cols; i++) {
    if (data[i] != zero) {
      return false;
    }
  }
  return true;
}

template <typename T>
__global__ void GetNonZeroNums(const T* dense_data,
                               const int rows,
                               const int cols,
                               int* non_zero_num,
                               int* temp_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ int counter;
  if (threadIdx.x == 0) counter = 0;
  __syncthreads();

  for (int i = tid; i < rows; i += gridDim.x * blockDim.x) {
    int index = -1;
    // TODO(zhangkaihuo): when cols=1, vectorization can be used
    if (!DevIsZero(dense_data + i * cols, cols)) {
      // use reductions?
      atomicAdd(&counter, 1);
      index = i;
    }
    temp_indexs[i] = index;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(non_zero_num, counter);
  }
}

template <typename T>
__global__ void GetNonZeroElementsAndIndices(const T* dense_data,
                                             const int64_t sparse_dim,
                                             const int64_t cols,
                                             const int64_t* x_dims,
                                             const int non_zero_num,
                                             const int* indexs,
                                             int64_t* indices,
                                             T* sparse_data) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int64_t sparse_index = indexs[i];
    int64_t x_index = sparse_index;
    for (int64_t j = sparse_dim - 1; j >= 0; j--) {
      indices[j * non_zero_num + i] = sparse_index % x_dims[j];
      sparse_index /= x_dims[j];
    }

    for (int j = 0; j < cols; j++) {
      sparse_data[i * cols + j] = dense_data[x_index * cols + j];
    }
  }
}

template <typename T, typename Context>
void DenseToSparseCooKernel(const Context& dev_ctx,
                            const DenseTensor& x,
                            const int64_t sparse_dim,
                            SparseCooTensor* out) {
  const T* x_data = x.data<T>();
  const auto& x_dims = x.dims();
  auto dims_2d = flatten_to_2d(x_dims, sparse_dim);
  const int rows = dims_2d[0];
  const int cols = dims_2d[1];
  auto nums_meta =
      phi::DenseTensorMeta(DataType::INT32, {1}, phi::DataLayout::NCHW);
  DenseTensor nums = phi::Empty(dev_ctx, std::move(nums_meta));
  auto x_dims_meta = phi::DenseTensorMeta(DataType::INT64,
                                          {static_cast<int64_t>(x_dims.size())},
                                          phi::DataLayout::NCHW);
  DenseTensor d_x_dims = phi::Empty(dev_ctx, std::move(x_dims_meta));

  const auto place = dev_ctx.GetPlace();

  // 1. get numbers of non zero elements, and get the index of non zero elements
  int* nums_ptr = nums.mutable_data<int>(place);
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(nums_ptr, 0, sizeof(int), dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(nums_ptr, 0, sizeof(int), dev_ctx.stream()));
#endif
  int grid_size = 1, block_size = 1;
  GetGpuLaunchConfig1D(dev_ctx, rows, &grid_size, &block_size);

  auto temp_indexs_meta =
      phi::DenseTensorMeta(DataType::INT32, {rows}, phi::DataLayout::NCHW);
  DenseTensor temp_indexs = phi::Empty(dev_ctx, std::move(temp_indexs_meta));
  int* temp_indexs_ptr = temp_indexs.mutable_data<int>(place);
  GetNonZeroNums<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
      x_data, rows, cols, nums_ptr, temp_indexs_ptr);
#ifdef PADDLE_WITH_HIP
  thrust::remove(thrust::hip::par.on(dev_ctx.stream()),
#else
  thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                 temp_indexs_ptr,
                 temp_indexs_ptr + rows,
                 -1);

  // 2. copy non_zero_num to host, copy x_dims to device
  int non_zero_num = 0;
#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&non_zero_num,
                                            nums_ptr,
                                            sizeof(int),
                                            hipMemcpyDeviceToHost,
                                            dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(&non_zero_num,
                                             nums_ptr,
                                             sizeof(int),
                                             hipMemcpyDeviceToHost,
                                             dev_ctx.stream()));
#endif

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemcpyAsync(d_x_dims.mutable_data<int64_t>(place),
                     x_dims.Get(),
                     x_dims.size() * sizeof(x_dims[0]),
                     hipMemcpyHostToDevice,
                     dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemcpyAsync(d_x_dims.mutable_data<int64_t>(place),
                      x_dims.Get(),
                      x_dims.size() * sizeof(x_dims[0]),
                      hipMemcpyHostToDevice,
                      dev_ctx.stream()));
#endif

  dev_ctx.Wait();  // wait the copy

  const auto values_dims = InferDenseDims(x_dims, sparse_dim, non_zero_num);
  DenseTensorMeta indices_meta(DataType::INT64,
                               {sparse_dim, static_cast<int64_t>(non_zero_num)},
                               DataLayout::NCHW);
  DenseTensorMeta values_meta(x.meta().dtype, values_dims, x.meta().layout);
  phi::DenseTensor indices(
      phi::make_intrusive<paddle::experimental::SharedStorage>(
          dev_ctx.GetPlace()),
      std::move(indices_meta));
  phi::DenseTensor values(
      phi::make_intrusive<paddle::experimental::SharedStorage>(
          dev_ctx.GetPlace()),
      std::move(values_meta));
  int64_t* indices_data = indices.mutable_data<int64_t>(place);
  T* sparse_data = values.mutable_data<T>(place);

  // 3. calc indices by indexs and get values by indexs
  GetGpuLaunchConfig1D(dev_ctx, non_zero_num, &grid_size, &block_size);
  GetNonZeroElementsAndIndices<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
      x_data,
      sparse_dim,
      cols,
      d_x_dims.data<int64_t>(),
      non_zero_num,
      temp_indexs_ptr,
      indices_data,
      sparse_data);
  out->SetMember(indices, values, x_dims, true);
}

__global__ void GetBatchSizes(const int64_t* crows,
                              const int rows,
                              const int batchs,
                              int* batch_sizes) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < batchs) {
    batch_sizes[tid] = crows[tid * (rows + 1) + rows];
  }
}

__global__ void ConvertCsrCrowsToCooRows(const int64_t* crows_ptr,
                                         const int* crows_offsets,
                                         int64_t* rows_ptr,
                                         int64_t* batch_ptr,
                                         const int rows) {
  const int b = blockIdx.y;
  const int64_t offset = crows_offsets ? crows_offsets[b] : 0;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < rows; i += gridDim.x * blockDim.x) {
    for (int j = crows_ptr[b * (rows + 1) + i];
         j < crows_ptr[b * (rows + 1) + i + 1];
         j++) {
      rows_ptr[offset + j] = i;
      if (batch_ptr) {
        batch_ptr[offset + j] = b;
      }
    }
  }
}

template <typename T, typename Context>
void SparseCsrToCooKernel(const Context& dev_ctx,
                          const SparseCsrTensor& x,
                          SparseCooTensor* out) {
  const DDim& x_dims = x.dims();
  const int64_t non_zero_num = x.non_zero_cols().numel();
  const auto& csr_crows = x.non_zero_crows();
  const auto& csr_cols = x.non_zero_cols();
  const auto& csr_values = x.non_zero_elements();
  const int64_t* csr_crows_data = csr_crows.data<int64_t>();
  const int64_t* csr_cols_data = csr_cols.data<int64_t>();
  const T* csr_values_data = csr_values.data<T>();

  int64_t sparse_dim = 2;
  if (x_dims.size() == 3) {
    sparse_dim = 3;
  }
  int batchs = x_dims.size() == 2 ? 1 : x_dims[0];
  int rows = x_dims.size() == 2 ? x_dims[0] : x_dims[1];

  const auto place = dev_ctx.GetPlace();
  DenseTensorMeta indices_meta(
      DataType::INT64, {sparse_dim, non_zero_num}, DataLayout::NCHW);
  DenseTensorMeta values_meta(x.dtype(), {non_zero_num}, x.layout());
  DenseTensorMeta offsets_meta(DataType::INT32, {batchs}, DataLayout::NCHW);
  DenseTensor indices = phi::Empty(dev_ctx, std::move(indices_meta));
  DenseTensor values = phi::Empty(dev_ctx, std::move(values_meta));
  DenseTensor offsets = phi::Empty(dev_ctx, std::move(offsets_meta));
  int64_t* coo_indices = indices.mutable_data<int64_t>(place);
  int64_t* batch_ptr = x_dims.size() == 2 ? nullptr : coo_indices;
  int64_t* coo_rows_data =
      x_dims.size() == 2 ? coo_indices : batch_ptr + non_zero_num;
  int64_t* coo_cols_data = coo_rows_data + non_zero_num;
  int* offsets_ptr = batchs == 1 ? nullptr : offsets.mutable_data<int>(place);
  T* coo_values_data = values.mutable_data<T>(place);

  int grid_size = 1, block_size = 1;
  if (batchs > 1) {
    GetGpuLaunchConfig1D(dev_ctx, batchs, &grid_size, &block_size);
    GetBatchSizes<<<grid_size, block_size>>>(
        csr_crows_data, rows, batchs, offsets_ptr);

#ifdef PADDLE_WITH_HIP
    thrust::exclusive_scan(thrust::hip::par.on(dev_ctx.stream()),
#else
    thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                           offsets_ptr,
                           offsets_ptr + batchs,
                           offsets_ptr);
  }

  GetGpuLaunchConfig1D(dev_ctx, rows, &grid_size, &block_size);
  dim3 grids(grid_size, batchs, 1);
  ConvertCsrCrowsToCooRows<<<grids, block_size>>>(
      csr_crows_data, offsets_ptr, coo_rows_data, batch_ptr, rows);

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_cols_data,
                                            csr_cols_data,
                                            sizeof(int64_t) * non_zero_num,
                                            hipMemcpyDeviceToDevice,
                                            dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_values_data,
                                            csr_values_data,
                                            sizeof(T) * non_zero_num,
                                            hipMemcpyDeviceToDevice,
                                            dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_cols_data,
                                             csr_cols_data,
                                             sizeof(int64_t) * non_zero_num,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(coo_values_data,
                                             csr_values_data,
                                             sizeof(T) * non_zero_num,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));
#endif

  out->SetMember(indices, values, x_dims, true);
}

__global__ void GetBatchsOffset(const int64_t* batchs_ptr,
                                const int non_zero_num,
                                int64_t* batchs_offset) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    if (i == non_zero_num - 1 || batchs_ptr[i] != batchs_ptr[i + 1]) {
      batchs_offset[batchs_ptr[i]] = i + 1;
    }
  }
}

__global__ void ConvertCooRowsToCsrCrows(
    const int64_t* batchs_offset,  // can be null if batchs = 1
    const int64_t* coo_rows_data,
    int64_t* csr_crows_data,
    const int rows,
    const int64_t non_zero_num) {
  const int b = blockIdx.y;
  int batch_non_zero_num =
      batchs_offset == nullptr ? non_zero_num : batchs_offset[b];
  if (batch_non_zero_num == 0) return;
  int batch_start = 0;
  if (b > 0) {
    batch_start = batchs_offset[b - 1];
    batch_non_zero_num -= batch_start;
  }
  auto* coo_rows_ptr = coo_rows_data + batch_start;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < batch_non_zero_num; i += gridDim.x * blockDim.x) {
    if (i == 0) {
      for (int j = 0; j <= coo_rows_ptr[0]; j++) {
        csr_crows_data[b * (rows + 1) + j] = 0;
      }
    } else {
      for (int j = coo_rows_ptr[i - 1]; j < coo_rows_ptr[i]; j++) {
        csr_crows_data[b * (rows + 1) + j + 1] = i;
      }
    }
    if (i == batch_non_zero_num - 1) {
      for (int64_t i = coo_rows_ptr[batch_non_zero_num - 1] + 1; i < rows + 1;
           i++) {
        csr_crows_data[b * (rows + 1) + i] = batch_non_zero_num;
      }
    }
  }
}

template <typename T, typename Context>
void SparseCooToCsrKernel(const Context& dev_ctx,
                          const SparseCooTensor& x,
                          SparseCsrTensor* out) {
  const auto& x_dims = x.dims();
  bool valid = x_dims.size() == 2 || x_dims.size() == 3;
  PADDLE_ENFORCE_EQ(valid,
                    true,
                    phi::errors::InvalidArgument(
                        "SparseCsrTensor only support 2-D or 3-D matrix"));
  const int64_t non_zero_num = x.nnz();
  if (non_zero_num <= 0) return;

  int batchs = x_dims.size() == 2 ? 1 : x_dims[0];
  int rows = x_dims.size() == 2 ? x_dims[0] : x_dims[1];

  const auto place = dev_ctx.GetPlace();
  DenseTensorMeta crows_meta(
      DataType::INT64, {batchs * (rows + 1)}, DataLayout::NCHW);
  DenseTensorMeta cols_meta(DataType::INT64, {non_zero_num}, DataLayout::NCHW);
  DenseTensorMeta values_meta(x.dtype(), {non_zero_num}, x.layout());
  phi::DenseTensor non_zero_crows(
      phi::make_intrusive<paddle::experimental::SharedStorage>(place),
      std::move(crows_meta));
  phi::DenseTensor non_zero_cols(
      phi::make_intrusive<paddle::experimental::SharedStorage>(place),
      std::move(cols_meta));
  phi::DenseTensor non_zero_elements(
      phi::make_intrusive<paddle::experimental::SharedStorage>(place),
      std::move(values_meta));
  int64_t* csr_crows_data = non_zero_crows.mutable_data<int64_t>(place);
  int64_t* csr_cols_data = non_zero_cols.mutable_data<int64_t>(place);
  T* csr_values_data = non_zero_elements.mutable_data<T>(place);

  const auto& coo_indices = x.non_zero_indices();
  const auto& coo_values = x.non_zero_elements();
  const int64_t* batchs_ptr = coo_indices.data<int64_t>();
  const int64_t* coo_rows_data =
      batchs == 1 ? batchs_ptr : batchs_ptr + non_zero_num;
  const int64_t* coo_cols_data = coo_rows_data + non_zero_num;
  const T* coo_values_data = coo_values.data<T>();

  if (!x.coalesced()) {
    // TODO(zhangkahuo): call coalesced() to distinct and sort the indices
  }

  int grid_size = 1, block_size = 1;
  GetGpuLaunchConfig1D(dev_ctx, batchs, &grid_size, &block_size);
  if (batchs > 1) {
    DenseTensorMeta batchs_meta(DataType::INT64, {batchs}, DataLayout::NCHW);
    phi::DenseTensor batchs_offset(
        phi::make_intrusive<paddle::experimental::SharedStorage>(place),
        std::move(batchs_meta));
    int64_t* batchs_offset_ptr = batchs_offset.mutable_data<int64_t>(place);
    GetBatchsOffset<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
        batchs_ptr, non_zero_num, batchs_offset_ptr);
    dim3 grids(grid_size, batchs, 1);
    ConvertCooRowsToCsrCrows<<<grids, block_size, 0, dev_ctx.stream()>>>(
        batchs_offset_ptr, coo_rows_data, csr_crows_data, rows, non_zero_num);
  } else {
    ConvertCooRowsToCsrCrows<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
        nullptr, coo_rows_data, csr_crows_data, rows, non_zero_num);
  }

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(csr_cols_data,
                                            coo_cols_data,
                                            sizeof(int64_t) * non_zero_num,
                                            hipMemcpyDeviceToDevice,
                                            dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(csr_values_data,
                                            coo_values_data,
                                            sizeof(T) * non_zero_num,
                                            hipMemcpyDeviceToDevice,
                                            dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(csr_cols_data,
                                             coo_cols_data,
                                             sizeof(int64_t) * non_zero_num,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(csr_values_data,
                                             coo_values_data,
                                             sizeof(T) * non_zero_num,
                                             hipMemcpyDeviceToDevice,
                                             dev_ctx.stream()));
#endif
  out->SetMember(non_zero_crows, non_zero_cols, non_zero_elements, x_dims);
}

template <typename ValueT, typename IndicesT>
__global__ void KernelSparseCooToDense(const IndicesT* indices,
                                       const IndicesT* sparse_offsets,
                                       const ValueT* data,
                                       ValueT* dense_data,
                                       const IndicesT non_zero_num,
                                       const int64_t base_offset,
                                       const int64_t sparse_dim) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int64_t index = 0;
    for (int j = 0; j < sparse_dim; j++) {
      index += indices[j * non_zero_num + i] * sparse_offsets[j];
    }

    for (int j = 0; j < base_offset; j++) {
      dense_data[index * base_offset + j] = data[i * base_offset + j];
    }
  }
}

template <typename T, typename Context>
void SparseCooToDenseKernel(const Context& dev_ctx,
                            const SparseCooTensor& x,
                            DenseTensor* out) {
  const auto non_zero_num = x.nnz();
  const auto dense_dims = x.dims();
  const auto indices = x.non_zero_indices();
  const auto values = x.non_zero_elements();
  const auto indices_dims = indices.dims();
  int64_t sparse_dim = indices_dims[0];
  if (indices_dims.size() == 1) {
    sparse_dim = 1;
  }
  const int64_t dense_dim = values.dims().size() - 1;

  const auto place = dev_ctx.GetPlace();
  const T* x_data = values.data<T>();
  T* out_data = out->mutable_data<T>(place);
  int64_t base_offset = 1;
  for (int64_t i = 0; i < dense_dim; i++) {
    base_offset *= dense_dims[sparse_dim + i];
  }
  std::vector<int64_t> sparse_offsets(sparse_dim);
  int64_t offset = 1;
  for (int i = sparse_dim - 1; i >= 0; i--) {
    sparse_offsets[i] = offset;
    offset *= dense_dims[i];
  }

  auto sparse_offset_meta = phi::DenseTensorMeta(
      DataType::INT64, {sparse_dim}, phi::DataLayout::NCHW);
  DenseTensor d_sparse_offsets = Empty(dev_ctx, std::move(sparse_offset_meta));

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemcpyAsync(d_sparse_offsets.mutable_data<int64_t>(place),
                     sparse_offsets.data(),
                     sparse_dim * sizeof(int64_t),
                     hipMemcpyHostToDevice,
                     dev_ctx.stream()));

  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(out_data, 0, sizeof(T) * out->numel(), dev_ctx.stream()));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemcpyAsync(d_sparse_offsets.mutable_data<int64_t>(place),
                      sparse_offsets.data(),
                      sparse_dim * sizeof(int64_t),
                      hipMemcpyHostToDevice,
                      dev_ctx.stream()));
  PADDLE_ENFORCE_GPU_SUCCESS(
      hipMemsetAsync(out_data, 0, sizeof(T) * out->numel(), dev_ctx.stream()));
#endif
  int grid_size = 1, block_size = 1;
  GetGpuLaunchConfig1D(dev_ctx, non_zero_num, &grid_size, &block_size);

  KernelSparseCooToDense<
      T,
      int64_t><<<grid_size, block_size, 0, dev_ctx.stream()>>>(
      indices.data<int64_t>(),
      d_sparse_offsets.data<int64_t>(),
      x_data,
      out_data,
      non_zero_num,
      base_offset,
      sparse_dim);
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(dense_to_sparse_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::DenseToSparseCooKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(sparse_csr_to_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SparseCsrToCooKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(sparse_coo_to_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SparseCooToCsrKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(dense_to_sparse_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::DenseToSparseCsrKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(sparse_coo_to_dense,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SparseCooToDenseKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(sparse_csr_to_dense,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SparseCsrToDenseKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
