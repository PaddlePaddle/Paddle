#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/sparse/sparse_utils_kernel.h"

#include <thrust/execution_policy.h>
#include <thrust/remove.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/sparse/common_shape.h"

namespace phi {
namespace sparse {

template <typename T>
inline __device__ bool DevIsZero(const T* data, const int64_t cols) {
  const T zero = static_cast<T>(0);
  // TODO(zhangkaihuo): check the data is zero or not in parallen when cols > 1
  for (int64_t i = 0; i < cols; i++) {
    if (data[i] != zero) {
      return false;
    }
  }
  return true;
}

template <typename T>
__global__ void GetNonZeroNums(const T* dense_data,
                               const int rows,
                               const int cols,
                               int* non_zero_num,
                               int* temp_indexs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ int counter;
  if (threadIdx.x == 0) counter = 0;
  __syncthreads();

  for (int i = tid; i < rows; i += gridDim.x * blockDim.x) {
    int index = -1;
    // TODO(zhangkaihuo): when cols=1, vectorization can be used
    if (!DevIsZero(dense_data + i * cols, cols)) {
      // use reductions?
      atomicAdd(&counter, 1);
      index = i;
    }
    temp_indexs[i] = index;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(non_zero_num, counter);
  }
}

template <typename T>
__global__ void GetNonZeroElementsAndIndices(const T* dense_data,
                                             const int64_t sparse_dim,
                                             const int64_t cols,
                                             const int64_t* x_dims,
                                             const int non_zero_num,
                                             const int* indexs,
                                             int64_t* indices,
                                             T* sparse_data) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int64_t sparse_index = indexs[i];
    int64_t x_index = sparse_index;
    for (int64_t j = sparse_dim - 1; j >= 0; j--) {
      indices[j * non_zero_num + i] = sparse_index % x_dims[j];
      sparse_index /= x_dims[j];
    }

    for (int j = 0; j < cols; j++) {
      sparse_data[i * cols + j] = dense_data[x_index * cols + j];
    }
  }
}

template <typename T, typename Context>
void DenseToCooKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      const int64_t sparse_dim,
                      SparseCooTensor* out) {
  const T* x_data = x.data<T>();
  const auto& x_dims = x.dims();
  PADDLE_ENFORCE_LE(sparse_dim,
                    x_dims.size(),
                    phi::errors::InvalidArgument(
                        "sparse_dim must be less than the size of x.dims()"));
  PADDLE_ENFORCE_GT(
      sparse_dim, 0, phi::errors::InvalidArgument("sparse_dim must be >0"));
  auto dims_2d = flatten_to_2d(x_dims, sparse_dim);
  const int rows = dims_2d[0];
  const int cols = dims_2d[1];
  DenseTensor nums = phi::Empty<int32_t>(dev_ctx, {1});
  DenseTensor d_x_dims = phi::Empty<int64_t>(dev_ctx, {x_dims.size()});

  // 1. get numbers of non zero elements, and get the index of non zero elements
  int* nums_ptr = nums.data<int>();
  phi::backends::gpu::GpuMemsetAsync(
      nums_ptr, 0, sizeof(int), dev_ctx.stream());
  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, rows, 1);

  DenseTensor temp_indexs = phi::Empty<int32_t>(dev_ctx, {rows});
  int* temp_indexs_ptr = temp_indexs.data<int>();

  GetNonZeroNums<<<config.block_per_grid.x,
                   config.thread_per_block.x,
                   0,
                   dev_ctx.stream()>>>(
      x_data, rows, cols, nums_ptr, temp_indexs_ptr);

#ifdef PADDLE_WITH_HIP
  thrust::remove(thrust::hip::par.on(dev_ctx.stream()),
#else
  thrust::remove(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                 temp_indexs_ptr,
                 temp_indexs_ptr + rows,
                 -1);

  // 2. copy non_zero_num to host, copy x_dims to device
  int non_zero_num = 0;
  phi::backends::gpu::GpuMemcpyAsync(&non_zero_num,
                                     nums_ptr,
                                     sizeof(int),
                                     gpuMemcpyDeviceToHost,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(d_x_dims.data<int64_t>(),
                                     x_dims.Get(),
                                     x_dims.size() * sizeof(x_dims[0]),
                                     gpuMemcpyHostToDevice,
                                     dev_ctx.stream());

  dev_ctx.Wait();  // wait the copy

  const auto values_dims =
      phi::funcs::sparse::InferDenseDims(x_dims, sparse_dim, non_zero_num);
  phi::DenseTensor indices = phi::Empty<int64_t>(
      dev_ctx, {sparse_dim, static_cast<int64_t>(non_zero_num)});
  int64_t* indices_data = indices.data<int64_t>();
  phi::DenseTensor values;
  values.Resize(values_dims);
  T* sparse_data = dev_ctx.template Alloc<T>(&values);

  // 3. calc indices by indexs and get values by indexs
  config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, non_zero_num, 1);
  GetNonZeroElementsAndIndices<<<config.block_per_grid.x,
                                 config.thread_per_block.x,
                                 0,
                                 dev_ctx.stream()>>>(x_data,
                                                     sparse_dim,
                                                     cols,
                                                     d_x_dims.data<int64_t>(),
                                                     non_zero_num,
                                                     temp_indexs_ptr,
                                                     indices_data,
                                                     sparse_data);

  out->SetMember(indices, values, x_dims, true);
}

template <typename IntT>
__global__ void GetBatchSizes(const IntT* crows,
                              const int rows,
                              const int batchs,
                              IntT* batch_sizes) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < batchs) {
    batch_sizes[tid] = crows[tid * (rows + 1) + rows];
  }
}

template <typename IntT>
__global__ void ConvertCsrCrowsToCooRows(const IntT* crows_ptr,
                                         const IntT* crows_offsets,
                                         IntT* rows_ptr,
                                         IntT* batch_ptr,
                                         const int rows) {
  const int b = blockIdx.y;
  const int64_t offset = crows_offsets ? crows_offsets[b] : 0;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < rows; i += gridDim.x * blockDim.x) {
    for (int j = crows_ptr[b * (rows + 1) + i];
         j < crows_ptr[b * (rows + 1) + i + 1];
         j++) {
      rows_ptr[offset + j] = i;
      if (batch_ptr) {
        batch_ptr[offset + j] = b;
      }
    }
  }
}

template <typename T, typename IntT>
void CsrToCooGPUKernel(const GPUContext& dev_ctx,
                       const SparseCsrTensor& x,
                       SparseCooTensor* out) {
  const DDim& x_dims = x.dims();
  const int64_t non_zero_num = x.cols().numel();
  const auto& csr_crows = x.crows();
  const auto& csr_cols = x.cols();
  const auto& csr_values = x.values();
  const IntT* csr_crows_data = csr_crows.data<IntT>();
  const IntT* csr_cols_data = csr_cols.data<IntT>();
  const T* csr_values_data = csr_values.data<T>();

  int64_t sparse_dim = 2;
  if (x_dims.size() == 3) {
    sparse_dim = 3;
  }
  int batchs = x_dims.size() == 2 ? 1 : x_dims[0];
  int rows = x_dims.size() == 2 ? x_dims[0] : x_dims[1];

  DenseTensor indices = phi::Empty<IntT>(dev_ctx, {sparse_dim, non_zero_num});
  DenseTensor values = phi::EmptyLike<T, GPUContext>(dev_ctx, csr_values);
  DenseTensor offsets = phi::Empty<IntT>(dev_ctx, {batchs});
  IntT* coo_indices = indices.data<IntT>();
  IntT* batch_ptr = x_dims.size() == 2 ? nullptr : coo_indices;
  IntT* coo_rows_data =
      x_dims.size() == 2 ? coo_indices : batch_ptr + non_zero_num;
  IntT* coo_cols_data = coo_rows_data + non_zero_num;
  IntT* offsets_ptr = batchs == 1 ? nullptr : offsets.data<IntT>();
  T* coo_values_data = values.data<T>();

  if (batchs > 1) {
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, batchs, 1);
    GetBatchSizes<IntT><<<config.block_per_grid.x, config.thread_per_block.x>>>(
        csr_crows_data, rows, batchs, offsets_ptr);

#ifdef PADDLE_WITH_HIP
    thrust::exclusive_scan(thrust::hip::par.on(dev_ctx.stream()),
#else
    thrust::exclusive_scan(thrust::cuda::par.on(dev_ctx.stream()),
#endif
                           offsets_ptr,
                           offsets_ptr + batchs,
                           offsets_ptr);
  }

  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, rows, 1);
  config.block_per_grid.y = batchs;
  ConvertCsrCrowsToCooRows<IntT>
      <<<config.block_per_grid, config.thread_per_block.x>>>(
          csr_crows_data, offsets_ptr, coo_rows_data, batch_ptr, rows);

  phi::backends::gpu::GpuMemcpyAsync(coo_cols_data,
                                     csr_cols_data,
                                     sizeof(IntT) * non_zero_num,
                                     gpuMemcpyDeviceToDevice,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(coo_values_data,
                                     csr_values_data,
                                     sizeof(T) * non_zero_num,
                                     gpuMemcpyDeviceToDevice,
                                     dev_ctx.stream());

  out->SetMember(indices, values, x_dims, true);
}

template <typename T, typename Context>
void CsrToCooKernel(const Context& dev_ctx,
                    const SparseCsrTensor& x,
                    SparseCooTensor* out) {
  PD_VISIT_BASE_INTEGRAL_TYPES(x.crows().dtype(), "CsrToCooGPUKernel", ([&] {
                                 CsrToCooGPUKernel<T, data_t>(dev_ctx, x, out);
                               }));
}

template <typename IntT>
__global__ void GetBatchsOffset(const IntT* batchs_ptr,
                                const int batchs,
                                const int non_zero_num,
                                int* batchs_offset) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    if (i == non_zero_num - 1 || batchs_ptr[i] != batchs_ptr[i + 1]) {
      const int start = batchs_ptr[i];
      const int end = i == non_zero_num - 1 ? batchs : batchs_ptr[i + 1];
      for (int j = start; j < end; j++) {
        batchs_offset[j] = i + 1;
      }
    }
  }
}

template <typename IntT>
__global__ void ConvertCooRowsToCsrCrows(
    const int* batchs_offset,  // can be null if batchs = 1
    const IntT* coo_rows_data,
    IntT* csr_crows_data,
    const int rows,
    const int64_t non_zero_num) {
  const int b = blockIdx.y;
  int batch_non_zero_num =
      batchs_offset == nullptr ? non_zero_num : batchs_offset[b];
  IntT batch_start = 0;
  if (b > 0) {
    batch_start = batchs_offset[b - 1];
    batch_non_zero_num -= batch_start;
  }

  const IntT* coo_rows_ptr = coo_rows_data + batch_start;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < batch_non_zero_num; i += gridDim.x * blockDim.x) {
    if (i == 0) {
      for (IntT j = 0; j <= coo_rows_ptr[0]; j++) {
        csr_crows_data[b * (rows + 1) + j] = 0;
      }
    } else {
      for (IntT j = coo_rows_ptr[i - 1]; j < coo_rows_ptr[i]; j++) {
        csr_crows_data[b * (rows + 1) + j + 1] = i;
      }
    }
    if (i == batch_non_zero_num - 1) {
      for (IntT i = coo_rows_ptr[batch_non_zero_num - 1] + 1; i < rows + 1;
           i++) {
        csr_crows_data[b * (rows + 1) + i] = batch_non_zero_num;
      }
    }
  }
  if (batch_non_zero_num == 0) {
    for (int i = tid; i < rows + 1; i += gridDim.x * blockDim.x) {
      csr_crows_data[b * (rows + 1) + i] = 0;
    }
  }
}

template <typename T, typename IntT>
void CooToCsrGPUKernel(const GPUContext& dev_ctx,
                       const SparseCooTensor& x,
                       SparseCsrTensor* out) {
  const auto& x_dims = x.dims();
  bool valid = x_dims.size() == 2 || x_dims.size() == 3;
  PADDLE_ENFORCE_EQ(valid,
                    true,
                    phi::errors::InvalidArgument(
                        "SparseCsrTensor only support 2-D or 3-D matrix"));
  const int64_t non_zero_num = x.nnz();
  if (non_zero_num <= 0) return;

  int batchs = x_dims.size() == 2 ? 1 : x_dims[0];
  int rows = x_dims.size() == 2 ? x_dims[0] : x_dims[1];

  phi::DenseTensor crows = phi::Empty<IntT>(dev_ctx, {batchs * (rows + 1)});
  phi::DenseTensor cols = phi::Empty<IntT>(dev_ctx, {non_zero_num});
  phi::DenseTensor values = phi::EmptyLike<T, GPUContext>(dev_ctx, x.values());
  IntT* csr_crows_data = crows.data<IntT>();
  IntT* csr_cols_data = cols.data<IntT>();
  T* csr_values_data = values.data<T>();

  const auto& coo_indices = x.indices();
  const auto& coo_values = x.values();
  const IntT* batchs_ptr = coo_indices.data<IntT>();
  const IntT* coo_rows_data =
      x_dims.size() == 2 ? batchs_ptr : batchs_ptr + non_zero_num;
  const IntT* coo_cols_data = coo_rows_data + non_zero_num;
  const T* coo_values_data = coo_values.data<T>();

  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, batchs, 1);
  if (batchs > 1) {
    auto config =
        phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, non_zero_num, 1);
    phi::DenseTensor batchs_offset = phi::Empty<int>(dev_ctx, {batchs});
    int* batchs_offset_ptr = batchs_offset.data<int>();
    phi::funcs::SetConstant<GPUContext, int> set_zero;
    // set zero if the nnz=0 of batchs[0]
    set_zero(dev_ctx, &batchs_offset, static_cast<IntT>(0));
    GetBatchsOffset<IntT><<<config.block_per_grid.x,
                            config.thread_per_block.x,
                            0,
                            dev_ctx.stream()>>>(
        batchs_ptr, batchs, non_zero_num, batchs_offset_ptr);

    config.block_per_grid.y = batchs;
    ConvertCooRowsToCsrCrows<IntT><<<config.block_per_grid,
                                     config.thread_per_block.x,
                                     0,
                                     dev_ctx.stream()>>>(
        batchs_offset_ptr, coo_rows_data, csr_crows_data, rows, non_zero_num);
  } else {
    ConvertCooRowsToCsrCrows<IntT><<<config.block_per_grid.x,
                                     config.thread_per_block.x,
                                     0,
                                     dev_ctx.stream()>>>(
        nullptr, coo_rows_data, csr_crows_data, rows, non_zero_num);
  }

  phi::backends::gpu::GpuMemcpyAsync(csr_cols_data,
                                     coo_cols_data,
                                     sizeof(IntT) * non_zero_num,
                                     gpuMemcpyDeviceToDevice,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemcpyAsync(csr_values_data,
                                     coo_values_data,
                                     sizeof(T) * non_zero_num,
                                     gpuMemcpyDeviceToDevice,
                                     dev_ctx.stream());
  out->SetMember(crows, cols, values, x_dims);
}

template <typename T, typename Context>
void CooToCsrKernel(const Context& dev_ctx,
                    const SparseCooTensor& x,
                    SparseCsrTensor* out) {
  PD_VISIT_BASE_INTEGRAL_TYPES(x.indices().dtype(), "CooToCsrGPUKernel", ([&] {
                                 CooToCsrGPUKernel<T, data_t>(dev_ctx, x, out);
                               }));
}

template <typename ValueT, typename IndicesT>
__global__ void KernelCooToDense(const IndicesT* indices,
                                 const int64_t* sparse_offsets,
                                 const ValueT* data,
                                 ValueT* dense_data,
                                 const IndicesT non_zero_num,
                                 const int64_t base_offset,
                                 const int64_t sparse_dim) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < non_zero_num; i += gridDim.x * blockDim.x) {
    int64_t index = 0;
    for (int j = 0; j < sparse_dim; j++) {
      index += indices[j * non_zero_num + i] * sparse_offsets[j];
    }

    for (int j = 0; j < base_offset; j++) {
      dense_data[index * base_offset + j] = data[i * base_offset + j];
    }
  }
}

template <typename T, typename IntT>
void CooToDenseGPUKernel(const GPUContext& dev_ctx,
                         const SparseCooTensor& x,
                         DenseTensor* out) {
  const auto non_zero_num = x.nnz();
  const auto dense_dims = x.dims();
  const auto indices = x.indices();
  const auto values = x.values();
  const auto indices_dims = indices.dims();
  int64_t sparse_dim = indices_dims[0];
  if (indices_dims.size() == 1) {
    sparse_dim = 1;
  }
  const int64_t dense_dim = values.dims().size() - 1;

  const auto place = dev_ctx.GetPlace();
  const T* x_data = values.data<T>();
  dev_ctx.template Alloc<T>(out);

  T* out_data = out->data<T>();
  int64_t base_offset = 1;
  for (int64_t i = 0; i < dense_dim; i++) {
    base_offset *= dense_dims[sparse_dim + i];
  }
  std::vector<int64_t> sparse_offsets(sparse_dim);
  int64_t offset = 1;
  for (int i = sparse_dim - 1; i >= 0; i--) {
    sparse_offsets[i] = offset;
    offset *= dense_dims[i];
  }

  DenseTensor d_sparse_offsets = Empty<int64_t>(dev_ctx, {sparse_dim});

  phi::backends::gpu::GpuMemcpyAsync(d_sparse_offsets.data<int64_t>(),
                                     sparse_offsets.data(),
                                     sparse_dim * sizeof(int64_t),
                                     gpuMemcpyHostToDevice,
                                     dev_ctx.stream());
  phi::backends::gpu::GpuMemsetAsync(
      out_data, 0, sizeof(T) * out->numel(), dev_ctx.stream());

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, non_zero_num, 1);

  KernelCooToDense<T, IntT>
      <<<config.block_per_grid.x,
         config.thread_per_block.x,
         0,
         dev_ctx.stream()>>>(indices.data<IntT>(),
                             d_sparse_offsets.data<int64_t>(),
                             x_data,
                             out_data,
                             non_zero_num,
                             base_offset,
                             sparse_dim);
}

template <typename T, typename Context>
void CooToDenseKernel(const Context& dev_ctx,
                      const SparseCooTensor& x,
                      DenseTensor* out) {
  PD_VISIT_BASE_INTEGRAL_TYPES(
      x.indices().dtype(), "CooToDenseGPUKernel", ([&] {
        CooToDenseGPUKernel<T, data_t>(dev_ctx, x, out);
      }));
}

}  // namespace sparse
}  // namespace phi

PD_REGISTER_KERNEL(dense_to_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::DenseToCooKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(csr_to_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CsrToCooKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t,
                   bool) {}

PD_REGISTER_KERNEL(coo_to_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CooToCsrKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t,
                   bool) {}

PD_REGISTER_KERNEL(dense_to_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::DenseToCsrKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(coo_to_dense,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CooToDenseKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(csr_to_dense,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::CsrToDenseKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(values_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::ValuesCooKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}

PD_REGISTER_KERNEL(values_csr,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::ValuesCsrKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_CSR);
}

PD_REGISTER_KERNEL(indices_coo,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::IndicesCooKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {
  kernel->InputAt(0).SetDataLayout(phi::DataLayout::SPARSE_COO);
}

PD_REGISTER_KERNEL(sparse_coo_tensor,
                   GPU,
                   ALL_LAYOUT,
                   phi::sparse::SparseCooTensorKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   uint8_t,
                   int16_t,
                   int,
                   int64_t) {}
