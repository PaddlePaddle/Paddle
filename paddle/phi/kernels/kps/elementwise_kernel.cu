#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#endif
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/elementwise_kernel_impl.h"

namespace phi {

// Create the definition of Maximum
DEFINE_CUDA_ELEMENTWISE_OP(Maximum)
template <typename T, typename Context>
void MaximumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MaximumRawKernel<T>(dev_ctx, x, y, axis, out);
}

// Create the definition of Minimum
DEFINE_CUDA_ELEMENTWISE_OP(Minimum)
template <typename T, typename Context>
void MinimumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MinimumRawKernel<T>(dev_ctx, x, y, axis, out);
}
// Create the definition of Remainder
DEFINE_CUDA_ELEMENTWISE_OP(Remainder)
// Create the definition of FloorDivide
DEFINE_CUDA_ELEMENTWISE_OP(FloorDivide)
template <typename T, typename Context>
void FloorDivideKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       DenseTensor* out) {
  int axis = -1;
  FloorDivideRawKernel<T>(dev_ctx, x, y, axis, out);
}
// Create the definition of Heaviside
DEFINE_CUDA_ELEMENTWISE_OP(ElementwiseHeaviside)
// Create the definition of Pow
DEFINE_CUDA_ELEMENTWISE_OP(ElementwisePow)
template <typename T, typename Context>
void ElementwisePowKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& y,
                          DenseTensor* out) {
  int axis = -1;
  ElementwisePowRawKernel<T>(dev_ctx, x, y, axis, out);
}

}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(maximum, KPS, ALL_LAYOUT, phi::MaximumKernel, float) {}
PD_REGISTER_KERNEL(maximum_raw, KPS, ALL_LAYOUT, phi::MaximumRawKernel, float) {
}
PD_REGISTER_KERNEL(minimum, KPS, ALL_LAYOUT, phi::MinimumKernel, float) {}
PD_REGISTER_KERNEL(minimum_raw, KPS, ALL_LAYOUT, phi::MinimumRawKernel, float) {
}
PD_REGISTER_KERNEL(floor_divide, KPS, ALL_LAYOUT, phi::FloorDivideKernel, int) {
}
PD_REGISTER_KERNEL(
    floor_divide_raw, KPS, ALL_LAYOUT, phi::FloorDivideRawKernel, int) {}
PD_REGISTER_KERNEL(
    elementwise_pow, KPS, ALL_LAYOUT, phi::ElementwisePowKernel, float) {}
PD_REGISTER_KERNEL(
    elementwise_pow_raw, KPS, ALL_LAYOUT, phi::ElementwisePowRawKernel, float) {
}

#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(fmax,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMaxKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}

PD_REGISTER_KERNEL(fmin,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMinKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}

PD_REGISTER_KERNEL(maximum_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::MaximumRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   bfloat16) {}
PD_REGISTER_KERNEL(minimum_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::MinimumRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   bfloat16) {}
PD_REGISTER_KERNEL(remainder_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::RemainderRawKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}
PD_REGISTER_KERNEL(floor_divide_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::FloorDivideRawKernel,
                   int,
                   int64_t) {}
PD_REGISTER_KERNEL(elementwise_heaviside_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::ElementwiseHeavisideRawKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}
PD_REGISTER_KERNEL(elementwise_pow_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::ElementwisePowRawKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}
#endif
