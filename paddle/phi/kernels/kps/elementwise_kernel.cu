#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#endif
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/elementwise_kernel_impl.h"
#include "paddle/phi/kernels/legacy/elementwise_kernel.h"

namespace phi {

template <typename T, typename Context>
void MaximumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MaximumRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void MinimumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MinimumRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void RemainderKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  int axis = -1;
  RemainderRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void FloorDivideKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       DenseTensor* out) {
  int axis = -1;
  FloorDivideRawKernel<T>(dev_ctx, x, y, axis, out);
}
// Create the definition of Heaviside
template <typename T, typename Context>
void HeavisideKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::ElementwiseHeavisideFunctor<T>());
}

template <typename T, typename Context>
void ElementwisePowKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& y,
                          DenseTensor* out) {
  int axis = -1;
  ElementwisePowRawKernel<T>(dev_ctx, x, y, axis, out);
}

}  // namespace phi

#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)

PD_REGISTER_KERNEL(maximum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MaximumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(minimum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MinimumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(remainder,
                   GPU,
                   ALL_LAYOUT,
                   phi::RemainderKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(floor_divide,
                   KPS,
                   ALL_LAYOUT,
                   phi::FloorDivideKernel,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(elementwise_pow,
                   KPS,
                   ALL_LAYOUT,
                   phi::ElementwisePowKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

#endif

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(maximum, KPS, ALL_LAYOUT, phi::MaximumKernel, float) {}
PD_REGISTER_KERNEL(minimum, KPS, ALL_LAYOUT, phi::MinimumKernel, float) {}
PD_REGISTER_KERNEL(floor_divide, KPS, ALL_LAYOUT, phi::FloorDivideKernel, int) {
}
PD_REGISTER_KERNEL(
    elementwise_pow, KPS, ALL_LAYOUT, phi::ElementwisePowKernel, float) {}

#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(fmax,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMaxKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(fmin,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMinKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(heaviside,
                   KPS,
                   ALL_LAYOUT,
                   phi::HeavisideKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}
#endif
