#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#endif
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/elementwise_add_kernel.h"
#include "paddle/phi/kernels/impl/elementwise_kernel_impl.h"
#include "paddle/phi/kernels/legacy/elementwise_kernel.h"

namespace phi {

template <typename T, typename Context>
void SubtractKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::SubtractFunctor<T>(), -1);
}

template <typename T, typename Context>
void MultiplyKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::MultiplyFunctor<T>(), -1);
}

template <typename T, typename Context>
void DivideKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const DenseTensor& y,
                  DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::DivideFunctor<T>(), -1);
}

template <typename T, typename Context>
void AddCudaFunctor(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    int axis,
                    DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::AddFunctor<T>(), axis);
}

template <typename T, typename Context>
void Float32Bfloat16OrFloat16AddCudaFunctor(const Context& dev_ctx,
                                            const DenseTensor& x,
                                            const DenseTensor& y,
                                            DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  if (y.dtype() == phi::DataType::BFLOAT16) {
    funcs::ElementwiseKernel<T>(
        dev_ctx, inputs, &outputs, funcs::Float32Bfloat16AddFunctor<T>());
  } else if (y.dtype() == phi::DataType::FLOAT16) {
    funcs::ElementwiseKernel<T>(
        dev_ctx, inputs, &outputs, funcs::Float32Float16AddFunctor<T>());
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Unsupport x dtype:%s, y dtype:%s for add(x, y) operation",
        phi::DataTypeToString(x.type()),
        phi::DataTypeToString(y.type())));
  }
}

template <typename T, typename Context>
void AddKernel(const Context& dev_ctx,
               const DenseTensor& x,
               const DenseTensor& y,
               DenseTensor* out) {
#ifdef PADDLE_WITH_CUDA
  if (x.dtype() == phi::DataType::FLOAT32 &&
      (y.dtype() == phi::DataType::BFLOAT16 ||
       y.dtype() == phi::DataType::FLOAT16)) {
    using Type = DataTypeToCppType<phi::DataType::FLOAT32>::type;
    Float32Bfloat16OrFloat16AddCudaFunctor<Type, Context>(dev_ctx, x, y, out);
  } else {
#endif
    AddCudaFunctor<T, Context>(dev_ctx, x, y, -1, out);
#ifdef PADDLE_WITH_CUDA
  }
#endif
}

template <typename T, typename Context>
void GradAddKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  AddCudaFunctor<T>(dev_ctx, x, y, -1, out);
}

template <typename T, typename Context>
void MaximumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MaximumRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void MinimumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MinimumRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void RemainderKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  int axis = -1;
  RemainderRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void FloorDivideKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       DenseTensor* out) {
  int axis = -1;
  FloorDivideRawKernel<T>(dev_ctx, x, y, axis, out);
}
// Create the definition of Heaviside
template <typename T, typename Context>
void HeavisideKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::ElementwiseHeavisideFunctor<T>());
}

template <typename T, typename Context>
void ElementwisePowKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& y,
                          DenseTensor* out) {
  int axis = -1;
  ElementwisePowRawKernel<T>(dev_ctx, x, y, axis, out);
}

}  // namespace phi

#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)

PD_REGISTER_KERNEL(maximum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MaximumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(minimum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MinimumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(remainder,
                   GPU,
                   ALL_LAYOUT,
                   phi::RemainderKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(floor_divide,
                   KPS,
                   ALL_LAYOUT,
                   phi::FloorDivideKernel,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(elementwise_pow,
                   KPS,
                   ALL_LAYOUT,
                   phi::ElementwisePowKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

#endif

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(maximum, KPS, ALL_LAYOUT, phi::MaximumKernel, float) {}
PD_REGISTER_KERNEL(minimum, KPS, ALL_LAYOUT, phi::MinimumKernel, float) {}
PD_REGISTER_KERNEL(divide, KPS, ALL_LAYOUT, phi::DivideKernel, float) {}
PD_REGISTER_KERNEL(multiply, KPS, ALL_LAYOUT, phi::MultiplyKernel, float) {}
PD_REGISTER_KERNEL(add, KPS, ALL_LAYOUT, phi::AddKernel, float) {}
PD_REGISTER_KERNEL(subtract, KPS, ALL_LAYOUT, phi::SubtractKernel, float) {}
PD_REGISTER_KERNEL(floor_divide, KPS, ALL_LAYOUT, phi::FloorDivideKernel, int) {
}
PD_REGISTER_KERNEL(
    elementwise_pow, KPS, ALL_LAYOUT, phi::ElementwisePowKernel, float) {}

#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(fmax,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMaxKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(fmin,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMinKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(heaviside,
                   KPS,
                   ALL_LAYOUT,
                   phi::HeavisideKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(add,
                   KPS,
                   ALL_LAYOUT,
                   phi::AddKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   complex64,
                   complex128) {}

PD_REGISTER_KERNEL(grad_add,
                   KPS,
                   ALL_LAYOUT,
                   phi::GradAddKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   complex64,
                   complex128) {}

PD_REGISTER_KERNEL(divide,
                   KPS,
                   ALL_LAYOUT,
                   phi::DivideKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}

PD_REGISTER_KERNEL(multiply,
                   KPS,
                   ALL_LAYOUT,
                   phi::MultiplyKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   bool,
                   float16,
                   complex64,
                   complex128,
                   bfloat16) {}

PD_REGISTER_KERNEL(subtract,
                   KPS,
                   ALL_LAYOUT,
                   phi::SubtractKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   int64_t,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}

#endif
