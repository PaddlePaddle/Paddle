#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#endif
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/elementwise_kernel_impl.h"
#include "paddle/phi/kernels/legacy/elementwise_add_kernel.h"
#include "paddle/phi/kernels/legacy/elementwise_divide_kernel.h"
#include "paddle/phi/kernels/legacy/elementwise_kernel.h"
#include "paddle/phi/kernels/legacy/elementwise_multipy_kernel.h"
#include "paddle/phi/kernels/legacy/elementwise_subtract_kernel.h"

namespace phi {

template <typename T, typename Context>
void SubtractKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    DenseTensor* out) {
  phi::SubtractRawKernel<T, Context>(dev_ctx, x, y, -1, out);
}

template <typename T, typename Context>
void MultiplyKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    DenseTensor* out) {
  phi::MultiplyRawKernel<T, Context>(dev_ctx, x, y, -1, out);
}

template <typename T, typename Context>
void DivideKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const DenseTensor& y,
                  DenseTensor* out) {
  phi::DivideRawKernel<T, Context>(dev_ctx, x, y, -1, out);
}

template <typename T, typename Context>
void MultiPrecisionAddKernelImpl(const Context& dev_ctx,
                                 const DenseTensor& x,
                                 const DenseTensor& y,
                                 DenseTensor* out) {
  std::vector<const DenseTensor*> inputs = {&x, &y};
  std::vector<DenseTensor*> outputs = {out};
  if (y.dtype() == phi::DataType::BFLOAT16) {
    funcs::ElementwiseKernel<T>(
        dev_ctx,
        inputs,
        &outputs,
        funcs::MultiPrecisionAddFunctor<T, phi::bfloat16>());
  } else if (y.dtype() == phi::DataType::FLOAT16) {
    funcs::ElementwiseKernel<T>(
        dev_ctx,
        inputs,
        &outputs,
        funcs::MultiPrecisionAddFunctor<T, phi::float16>());
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Unsupport x dtype:%s, y dtype:%s for add(x, y) operation",
        phi::DataTypeToString(x.type()),
        phi::DataTypeToString(y.type())));
  }
}

template <typename T, typename Context>
void AddKernel(const Context& dev_ctx,
               const DenseTensor& x,
               const DenseTensor& y,
               DenseTensor* out) {
#ifdef PADDLE_WITH_CUDA
  if (x.dtype() == phi::DataType::FLOAT32 &&
      (y.dtype() == phi::DataType::BFLOAT16 ||
       y.dtype() == phi::DataType::FLOAT16)) {
    MultiPrecisionAddKernelImpl<float, Context>(dev_ctx, x, y, out);
  } else {
#endif
    phi::AddRawKernel<T, Context>(dev_ctx, x, y, -1, out);
#ifdef PADDLE_WITH_CUDA
  }
#endif
}

template <typename T, typename Context>
void GradAddKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  phi::AddRawKernel<T>(dev_ctx, x, y, -1, out);
}

template <typename T, typename Context>
void MaximumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MaximumRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void MinimumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  int axis = -1;
  MinimumRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void RemainderKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  int axis = -1;
  RemainderRawKernel<T>(dev_ctx, x, y, axis, out);
}

template <typename T, typename Context>
void FloorDivideKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       DenseTensor* out) {
  int axis = -1;
  FloorDivideRawKernel<T>(dev_ctx, x, y, axis, out);
}

// Create the definition of Heaviside
template <typename T, typename Context>
void HeavisideKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  std::vector<const DenseTensor*> inputs = {&x, &y};
  std::vector<DenseTensor*> outputs = {out};
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<T>(
      dev_ctx, inputs, &outputs, funcs::ElementwiseHeavisideFunctor<T>());
}

template <typename T, typename Context>
void ElementwisePowKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& y,
                          DenseTensor* out) {
  int axis = -1;
  ElementwisePowRawKernel<T>(dev_ctx, x, y, axis, out);
}

}  // namespace phi

#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)

PD_REGISTER_KERNEL(maximum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MaximumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(minimum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MinimumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(remainder,
                   GPU,
                   ALL_LAYOUT,
                   phi::RemainderKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(floor_divide,
                   KPS,
                   ALL_LAYOUT,
                   phi::FloorDivideKernel,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(elementwise_pow,
                   KPS,
                   ALL_LAYOUT,
                   phi::ElementwisePowKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

#endif

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(maximum, KPS, ALL_LAYOUT, phi::MaximumKernel, float) {}
PD_REGISTER_KERNEL(minimum, KPS, ALL_LAYOUT, phi::MinimumKernel, float) {}
PD_REGISTER_KERNEL(divide, KPS, ALL_LAYOUT, phi::DivideKernel, float) {}
PD_REGISTER_KERNEL(multiply, KPS, ALL_LAYOUT, phi::MultiplyKernel, float) {}
PD_REGISTER_KERNEL(add, KPS, ALL_LAYOUT, phi::AddKernel, float) {}
PD_REGISTER_KERNEL(subtract, KPS, ALL_LAYOUT, phi::SubtractKernel, float) {}
PD_REGISTER_KERNEL(floor_divide, KPS, ALL_LAYOUT, phi::FloorDivideKernel, int) {
}
PD_REGISTER_KERNEL(
    elementwise_pow, KPS, ALL_LAYOUT, phi::ElementwisePowKernel, float) {}

#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(fmax,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMaxKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(fmin,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMinKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(heaviside,
                   KPS,
                   ALL_LAYOUT,
                   phi::HeavisideKernel,
                   float,
                   double,
                   int,
                   float16,
                   bfloat16,
                   int64_t) {}

PD_REGISTER_KERNEL(add,
                   KPS,
                   ALL_LAYOUT,
                   phi::AddKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   bool,
                   uint8_t,
                   int8_t,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   complex64,
                   complex128) {}

PD_REGISTER_KERNEL(grad_add,
                   KPS,
                   ALL_LAYOUT,
                   phi::GradAddKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   bool,
                   uint8_t,
                   int8_t,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   complex64,
                   complex128) {}

PD_REGISTER_KERNEL(divide,
                   KPS,
                   ALL_LAYOUT,
                   phi::DivideKernel,
                   float,
                   double,
                   int8_t,
                   uint8_t,
                   int16_t,
                   int,
                   int64_t,
                   bool,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}

PD_REGISTER_KERNEL(multiply,
                   KPS,
                   ALL_LAYOUT,
                   phi::MultiplyKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   bool,
                   float16,
                   complex64,
                   complex128,
                   bfloat16) {}

PD_REGISTER_KERNEL(subtract,
                   KPS,
                   ALL_LAYOUT,
                   phi::SubtractKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   int64_t,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}

#endif
