#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#endif
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/elementwise_kernel_impl.h"

namespace phi {

template <typename T, typename Context>
void MaximumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(
      dev_ctx, inputs, &outputs, -1, funcs::MaximumFunctor<T>());
}

template <typename T, typename Context>
void MinimumKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(
      dev_ctx, inputs, &outputs, -1, funcs::MinimumFunctor<T>());
}

template <typename T, typename Context>
void RemainderKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(
      dev_ctx, inputs, &outputs, -1, funcs::RemainderFunctor<T>());
}

template <typename T, typename Context>
void FloorDivideKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(
      dev_ctx, inputs, &outputs, -1, funcs::FloorDivideFunctor<T>());
}
// Create the definition of Heaviside
template <typename T, typename Context>
void HeavisideKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(
      dev_ctx, inputs, &outputs, -1, funcs::ElementwiseHeavisideFunctor<T>());
}

template <typename T, typename Context>
void ElementwisePowKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& y,
                          DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  inputs.reserve(2);
  std::vector<DenseTensor*> outputs;
  outputs.reserve(1);
  inputs.emplace_back(&x);
  inputs.emplace_back(&y);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(
      dev_ctx, inputs, &outputs, -1, funcs::ElementwisePowFunctor<T>());
}

}  // namespace phi

#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)

PD_REGISTER_KERNEL(maximum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MaximumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(minimum,
                   KPS,
                   ALL_LAYOUT,
                   phi::MinimumKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(remainder,
                   GPU,
                   ALL_LAYOUT,
                   phi::RemainderKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(
    floor_divide, KPS, ALL_LAYOUT, phi::FloorDivideKernel, int, int64_t) {}
PD_REGISTER_KERNEL(elementwise_pow,
                   KPS,
                   ALL_LAYOUT,
                   phi::ElementwisePowKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

#endif

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(maximum, KPS, ALL_LAYOUT, phi::MaximumKernel, float) {}
PD_REGISTER_KERNEL(minimum, KPS, ALL_LAYOUT, phi::MinimumKernel, float) {}
PD_REGISTER_KERNEL(floor_divide, KPS, ALL_LAYOUT, phi::FloorDivideKernel, int) {
}
PD_REGISTER_KERNEL(
    elementwise_pow, KPS, ALL_LAYOUT, phi::ElementwisePowKernel, float) {}

#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(fmax,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMaxKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}

PD_REGISTER_KERNEL(fmin,
                   KPS,
                   ALL_LAYOUT,
                   phi::FMinKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}

PD_REGISTER_KERNEL(heaviside,
                   KPS,
                   ALL_LAYOUT,
                   phi::HeavisideKernel,
                   float,
                   double,
                   int,
                   float16,
                   int64_t) {}
#endif
