// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/impl/compare_kernel_impl.h"

#ifdef PADDLE_WITH_XPU_KP
#include "paddle/phi/backends/xpu/xpu_context.h"
#else
#include <thrust/fill.h>

#include <vector>

#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/kernels/compare_kernel.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"
#endif

namespace phi {

template <typename T>
struct BitwiseAdd {
  // Bitwise add operator, returns <tt>a + b</tt>
  inline T initial() { return static_cast<T>(true); }

  __host__ __device__ __forceinline__ T operator()(const T& a,
                                                   const T& b) const {
    return a & b;
  }
};

template <typename T,
          typename Context,
          typename Functor,
          typename InverseFunctor>
inline void CompareKernelImpl(const Context& ctx,
                              const DenseTensor& x,
                              const DenseTensor& y,
                              int axis,
                              DenseTensor* out) {
  ctx.template Alloc<bool>(out);
  std::vector<const DenseTensor*> ins{&x, &y};
  std::vector<DenseTensor*> outs{out};
  funcs::BroadcastKernel<ElementwiseType::kBinary, T, bool>(
      ctx, ins, &outs, axis, Functor());
}

#ifndef PADDLE_WITH_XPU_KP
template <typename T, typename Context, typename Functor>
inline void CompareAllKernelImpl(const Context& ctx,
                                 const DenseTensor& x,
                                 const DenseTensor& y,
                                 DenseTensor* out) {
  bool* out_data = ctx.template Alloc<bool>(out);

  if (x.dims() != y.dims()) {
    thrust::device_ptr<bool> out_dev_ptr(out_data);
    thrust::fill(out_dev_ptr, out_dev_ptr + 1, false);
    return;
  }

  DenseTensor tmp;
  tmp.Resize(x.dims());
  ctx.template Alloc<bool>(&tmp);

  std::vector<const DenseTensor*> ins{&x, &y};
  std::vector<DenseTensor*> outs{&tmp};
  funcs::ElementwiseKernel<bool>(ctx, ins, &outs, Functor());

  // Reduce by 'bitwise and' operator
  std::vector<int> reduce_dims;
  reduce_dims.resize(tmp.dims().size());
  for (int i = 0; i < reduce_dims.size(); ++i) {
    reduce_dims[i] = i;
  }
  funcs::ReduceKernel<bool, bool, BitwiseAdd, kps::IdentityFunctor<bool>>(
      ctx, tmp, out, kps::IdentityFunctor<bool>(), reduce_dims);
}
#endif

}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(less_than, KPS, ALL_LAYOUT, phi::LessThanKernel, int) {}
PD_REGISTER_KERNEL(less_equal, KPS, ALL_LAYOUT, phi::LessEqualKernel, int) {}
PD_REGISTER_KERNEL(greater_than, KPS, ALL_LAYOUT, phi::GreaterThanKernel, int) {
}
PD_REGISTER_KERNEL(
    greater_equal, KPS, ALL_LAYOUT, phi::GreaterEqualKernel, int) {}
PD_REGISTER_KERNEL(equal, KPS, ALL_LAYOUT, phi::EqualKernel, int) {}
PD_REGISTER_KERNEL(not_equal, KPS, ALL_LAYOUT, phi::NotEqualKernel, int) {}
#else
PD_REGISTER_KERNEL(less_than,
                   KPS,
                   ALL_LAYOUT,
                   phi::LessThanKernel,
                   bool,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(less_equal,
                   KPS,
                   ALL_LAYOUT,
                   phi::LessEqualKernel,
                   bool,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(greater_than,
                   KPS,
                   ALL_LAYOUT,
                   phi::GreaterThanKernel,
                   bool,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(greater_equal,
                   KPS,
                   ALL_LAYOUT,
                   phi::GreaterEqualKernel,
                   bool,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(equal,
                   KPS,
                   ALL_LAYOUT,
                   phi::EqualKernel,
                   bool,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(not_equal,
                   KPS,
                   ALL_LAYOUT,
                   phi::NotEqualKernel,
                   bool,
                   int16_t,
                   int,
                   int64_t,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(equal_all,
                   KPS,
                   ALL_LAYOUT,
                   phi::EqualAllKernel,
                   bool,
                   int,
                   int64_t,
                   float,
                   double) {}
#endif
