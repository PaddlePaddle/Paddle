#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <limits>
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/prod_kernel.h"
#include "paddle/phi/kernels/reduce_all_kernel.h"
#include "paddle/phi/kernels/reduce_amin_kernel.h"
#include "paddle/phi/kernels/reduce_any_kernel.h"
#include "paddle/phi/kernels/reduce_max_kernel.h"
#include "paddle/phi/kernels/reduce_mean_kernel.h"
#include "paddle/phi/kernels/reduce_min_kernel.h"
#include "paddle/phi/kernels/reduce_sum_kernel.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/kernels/funcs/eigen/common.h"
#endif

namespace phi {

template <typename T, typename Context>
void ProdKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const IntArray& dims,
                bool keep_dim,
                bool reduce_all,
                DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = x.dtype();
  phi::Reduce<T, kps::MulFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void AllRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const std::vector<int64_t>& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = phi::DataType::BOOL;
  phi::Reduce<T, kps::LogicalAndFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void AMaxRawKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const std::vector<int64_t>& dims,
                   bool keep_dim,
                   bool reduce_all,
                   DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = x.dtype();
  phi::Reduce<T, kps::MaxFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void AMinRawKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const std::vector<int64_t>& dims,
                   bool keep_dim,
                   bool reduce_all,
                   DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = x.dtype();
  phi::Reduce<T, kps::MinFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void AnyRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const std::vector<int64_t>& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = phi::DataType::BOOL;
  phi::Reduce<T, kps::LogicalOrFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void MaxKernel(const Context& dev_ctx,
               const DenseTensor& x,
               const IntArray& dims,
               bool keep_dim,
               DenseTensor* out) {
  bool reduce_all = recompute_reduce_all(x, dims);
  auto out_dtype = x.dtype();
  phi::Reduce<T, kps::MaxFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out);
}

template <typename T, typename Context>
void MeanRawKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const IntArray& dims,
                   bool keep_dim,
                   bool reduce_all,
                   DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = x.dtype();
  phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out, true);
}

template <typename T, typename Context>
void MinRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const IntArray& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = x.dtype();
  phi::Reduce<T, kps::MinFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out);
}

#ifndef PADDLE_WITH_XPU_KP
template <typename T,
          int EigenDimSize = 5,
          int ReducedDimSize = 1,
          bool ReduceAll = false>
void ReduceSumEigen(const KPDevice& dev_ctx,
                    const DenseTensor& x,
                    bool reduce_all,
                    const std::vector<int64_t>& dims,
                    DataType out_dtype,
                    DenseTensor* out,
                    std::vector<int>* reduce_dims) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  // Resize Input Tensor
  auto new_x = x;
  int added_dims = EigenDimSize - x.dims().size();
  std::array<int64_t, EigenDimSize> new_x_dim;
  new_x_dim.fill(1);
  for (int i = 0; i < x.dims().size(); i++) {
    new_x_dim[i + added_dims] = x.dims().at(i);
  }
  new_x.Resize(phi::DDim(new_x_dim.data(), new_x_dim.size()));
  auto eigen_x_tensor = EigenTensor<T, EigenDimSize>::From(new_x);

  // Create Out Tensor
  dev_ctx.Alloc<T>(out);
  auto origin_out_dims = out->dims();
  constexpr int kReduceOutRank = ReduceAll ? 1 : EigenDimSize - ReducedDimSize;
  // Resize Out Tensor
  std::array<int64_t, kReduceOutRank> new_out_dim;
  new_out_dim.fill(1);
  for (int i = 0; i < out->dims().size(); i++) {
    new_out_dim[i + added_dims] = out->dims().at(i);
  }
  out->Resize(phi::DDim(new_out_dim.data(), new_out_dim.size()));

  auto eigen_out_tensor = EigenTensor<T, kReduceOutRank>::From(*out);
  for (int i = 0; i < ReducedDimSize; i++) {
    (*reduce_dims)[i] += added_dims;
  }
  auto eigen_reduce_dim =
      EigenDim<ReducedDimSize>::From(phi::make_ddim(*reduce_dims));
  // Caculate
  eigen_out_tensor.device(*dev_ctx.eigen_device()) =
      eigen_x_tensor.sum(eigen_reduce_dim);
  out->Resize(origin_out_dims);
}
#endif

template <typename T, typename Context>
void SumRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const IntArray& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DataType out_dtype,
                  DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  if (out_dtype == DataType::UNDEFINED && out->dtype() != x.dtype()) {
    out_dtype = out->dtype();
  }
  if (x.numel() > std::numeric_limits<int32_t>::max()) {
#ifndef PADDLE_WITH_XPU_KP
    if (out_dtype != phi::DataType::UNDEFINED && out_dtype != x.dtype()) {
      PADDLE_THROW(phi::errors::Fatal(
          "If Input.numel() > INT32_MAX, reduce_sum kernel uses EigenTensor "
          "sum for reduce_sum function. As a result, input dtype should be "
          "the same as out dtype"));
    }

    std::vector<int> reduce_dims = phi::funcs::details::GetReduceDim(
        dims.GetData(), x.dims().size(), reduce_all);

#define CALL_EIGEN_REDUCE_SUM_KERNEL(reduce_rank)              \
  case reduce_rank: {                                          \
    if (reduce_all) {                                          \
      ReduceSumEigen<T, 5, reduce_rank, true>(dev_ctx,         \
                                              x,               \
                                              reduce_all,      \
                                              dims.GetData(),  \
                                              out_dtype,       \
                                              out,             \
                                              &reduce_dims);   \
    } else {                                                   \
      ReduceSumEigen<T, 5, reduce_rank, false>(dev_ctx,        \
                                               x,              \
                                               reduce_all,     \
                                               dims.GetData(), \
                                               out_dtype,      \
                                               out,            \
                                               &reduce_dims);  \
    }                                                          \
    break;                                                     \
  }

    switch (reduce_dims.size()) {
      CALL_EIGEN_REDUCE_SUM_KERNEL(1);
      CALL_EIGEN_REDUCE_SUM_KERNEL(2);
      CALL_EIGEN_REDUCE_SUM_KERNEL(3);
      CALL_EIGEN_REDUCE_SUM_KERNEL(4);
      CALL_EIGEN_REDUCE_SUM_KERNEL(5);
      default:
        PADDLE_THROW(phi::errors::Fatal(
            "If Input.numel() > INT32_MAX, reduce_sum kernel uses EigenTensor "
            "sum for reduce_sum function. As a result, its dim should be <= "
            "5."));
        break;
    }
#undef CALL_EIGEN_REDUCE_SUM_KERNEL
#else
    PADDLE_THROW(phi::errors::Fatal(
        "If Input.numel() > INT32_MAX, reduce_sum kernel uses EigenTensor "
        "sum for reduce_sum function. Such case is only supported on GPU "
        "now."));
#endif
  } else {
    phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
        dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out);
  }
}
}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(all_raw, KPS, ALL_LAYOUT, phi::AllRawKernel, bool) {
  kernel->OutputAt(0).SetDataType(phi::DataType::BOOL);
}

PD_REGISTER_KERNEL(amax_raw, KPS, ALL_LAYOUT, phi::AMaxRawKernel, float) {}

PD_REGISTER_KERNEL(prod, KPS, ALL_LAYOUT, phi::ProdKernel, float) {}

PD_REGISTER_KERNEL(amin_raw, KPS, ALL_LAYOUT, phi::AMinRawKernel, float) {}

PD_REGISTER_KERNEL(any_raw, KPS, ALL_LAYOUT, phi::AnyRawKernel, bool) {}

PD_REGISTER_KERNEL(max, KPS, ALL_LAYOUT, phi::MaxKernel, float) {}

PD_REGISTER_KERNEL(mean_raw, KPS, ALL_LAYOUT, phi::MeanRawKernel, float) {}

PD_REGISTER_KERNEL(min_raw, KPS, ALL_LAYOUT, phi::MinRawKernel, float) {}

PD_REGISTER_KERNEL(sum_raw, KPS, ALL_LAYOUT, phi::SumRawKernel, float) {
  kernel->OutputAt(0).SetDataType(phi::DataType::UNDEFINED);
}
#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(all_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::AllRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   bool) {
  kernel->OutputAt(0).SetDataType(phi::DataType::BOOL);
}

PD_REGISTER_KERNEL(amax_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::AMaxRawKernel,
                   float,
                   double,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(amin_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::AMinRawKernel,
                   float,
                   double,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(any_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::AnyRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   bool,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->OutputAt(0).SetDataType(phi::DataType::BOOL);
}

PD_REGISTER_KERNEL(max,
                   KPS,
                   ALL_LAYOUT,
                   phi::MaxKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(mean_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::MeanRawKernel,
                   float,
                   double,
                   bool,
                   phi::dtype::bfloat16,
                   float16,
                   int,
                   int64_t,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(min_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::MinRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(sum_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::SumRawKernel,
                   bool,
                   float,
                   double,
                   float16,
                   bfloat16,
                   int16_t,
                   int,
                   int64_t,
                   complex64,
                   complex128) {
  kernel->OutputAt(0).SetDataType(phi::DataType::UNDEFINED);
}

PD_REGISTER_KERNEL(prod,
                   KPS,
                   ALL_LAYOUT,
                   phi::ProdKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#endif
