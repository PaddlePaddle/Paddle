// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/reduce_sum_kernel.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/reduce.h"

namespace phi {

template <typename T, typename Context>
void SumRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const std::vector<int64_t>& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DataType out_dtype,
                  DenseTensor* out) {
  if (out_dtype == DataType::UNDEFINED && out->dtype() != x.dtype()) {
    out_dtype = out->dtype();
  }
  phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims, keep_dim, out_dtype, out);
}

}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(sum_raw, KPS, ALL_LAYOUT, phi::SumRawKernel, float) {
  kernel->OutputAt(0).SetDataType(paddle::experimental::DataType::UNDEFINED);
}
#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(sum_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::SumRawKernel,
                   bool,
                   float,
                   double,
                   float16,
                   bfloat16,
                   int16_t,
                   int,
                   int64_t,
                   complex64,
                   complex128) {
  kernel->OutputAt(0).SetDataType(paddle::experimental::DataType::UNDEFINED);
}
#endif
