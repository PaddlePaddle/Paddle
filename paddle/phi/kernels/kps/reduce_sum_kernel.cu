#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/reduce_sum_kernel.h"
#include <limits>
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#ifndef PADDLE_WITH_XPU_KP
#include "paddle/phi/kernels/funcs/eigen/common.h"
#endif

namespace phi {

#ifndef PADDLE_WITH_XPU_KP
template <typename T,
          int EigenDimSize = 5,
          int ReducedDimSize = 1,
          bool ReduceAll = false>
void ReduceSumEigen(const KPDevice& dev_ctx,
                    const DenseTensor& x,
                    bool reduce_all,
                    const std::vector<int64_t>& dims,
                    DataType out_dtype,
                    DenseTensor* out,
                    std::vector<int>* reduce_dims) {
  // Resize Input Tensor
  auto new_x = x;
  int added_dims = EigenDimSize - x.dims().size();
  std::array<int64_t, EigenDimSize> new_x_dim;
  new_x_dim.fill(1);
  for (int i = 0; i < x.dims().size(); i++) {
    new_x_dim[i + added_dims] = x.dims().at(i);
  }
  new_x.Resize(phi::DDim(new_x_dim.data(), new_x_dim.size()));
  auto eigen_x_tensor = EigenTensor<T, EigenDimSize>::From(new_x);

  // Create Out Tensor
  dev_ctx.Alloc<T>(out);
  auto origin_out_dims = out->dims();
  constexpr int kReduceOutRank = ReduceAll ? 1 : EigenDimSize - ReducedDimSize;
  // Resize Out Tensor
  std::array<int64_t, kReduceOutRank> new_out_dim;
  new_out_dim.fill(1);
  for (int i = 0; i < out->dims().size(); i++) {
    new_out_dim[i + added_dims] = out->dims().at(i);
  }
  out->Resize(phi::DDim(new_out_dim.data(), new_out_dim.size()));

  auto eigen_out_tensor = EigenTensor<T, kReduceOutRank>::From(*out);
  for (int i = 0; i < ReducedDimSize; i++) {
    (*reduce_dims)[i] += added_dims;
  }
  auto eigen_reduce_dim =
      EigenDim<ReducedDimSize>::From(phi::make_ddim(*reduce_dims));
  // Caculate
  eigen_out_tensor.device(*dev_ctx.eigen_device()) =
      eigen_x_tensor.sum(eigen_reduce_dim);
  out->Resize(origin_out_dims);
}
#endif

template <typename T, typename Context>
void SumRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const IntArray& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DataType out_dtype,
                  DenseTensor* out) {
  if (out_dtype == DataType::UNDEFINED && out->dtype() != x.dtype()) {
    out_dtype = out->dtype();
  }
  if (x.numel() > std::numeric_limits<int32_t>::max()) {
#ifndef PADDLE_WITH_XPU_KP
    if (out_dtype != phi::DataType::UNDEFINED && out_dtype != x.dtype()) {
      PADDLE_THROW(phi::errors::Fatal(
          "If Input.numel() > INT32_MAX, reduce_sum kernel uses EigenTensor "
          "sum for reduce_sum function. As a result, input dtype should be "
          "the same as out dtype"));
    }

    std::vector<int> reduce_dims = phi::funcs::details::GetReduceDim(
        dims.GetData(), x.dims().size(), reduce_all);

#define CALL_EIGEN_REDUCE_SUM_KERNEL(reduce_rank)              \
  case reduce_rank: {                                          \
    if (reduce_all) {                                          \
      ReduceSumEigen<T, 5, reduce_rank, true>(dev_ctx,         \
                                              x,               \
                                              reduce_all,      \
                                              dims.GetData(),  \
                                              out_dtype,       \
                                              out,             \
                                              &reduce_dims);   \
    } else {                                                   \
      ReduceSumEigen<T, 5, reduce_rank, false>(dev_ctx,        \
                                               x,              \
                                               reduce_all,     \
                                               dims.GetData(), \
                                               out_dtype,      \
                                               out,            \
                                               &reduce_dims);  \
    }                                                          \
    break;                                                     \
  }

    switch (reduce_dims.size()) {
      CALL_EIGEN_REDUCE_SUM_KERNEL(1);
      CALL_EIGEN_REDUCE_SUM_KERNEL(2);
      CALL_EIGEN_REDUCE_SUM_KERNEL(3);
      CALL_EIGEN_REDUCE_SUM_KERNEL(4);
      CALL_EIGEN_REDUCE_SUM_KERNEL(5);
      default:
        PADDLE_THROW(phi::errors::Fatal(
            "If Input.numel() > INT32_MAX, reduce_sum kernel uses EigenTensor "
            "sum for reduce_sum function. As a result, its dim should be <= "
            "5."));
        break;
    }
#undef CALL_EIGEN_REDUCE_SUM_KERNEL
#else
    PADDLE_THROW(phi::errors::Fatal(
        "If Input.numel() > INT32_MAX, reduce_sum kernel uses EigenTensor "
        "sum for reduce_sum function. Such case is only supported on GPU "
        "now."));
#endif
  } else {
    phi::Reduce<T, kps::AddFunctor, kps::IdentityFunctor>(
        dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out);
  }
}
}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(sum_raw, KPS, ALL_LAYOUT, phi::SumRawKernel, float) {
  kernel->OutputAt(0).SetDataType(paddle::experimental::DataType::UNDEFINED);
}
#else
using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(sum_raw,
                   KPS,
                   ALL_LAYOUT,
                   phi::SumRawKernel,
                   bool,
                   float,
                   double,
                   float16,
                   bfloat16,
                   int16_t,
                   int,
                   int64_t,
                   complex64,
                   complex128) {
  kernel->OutputAt(0).SetDataType(paddle::experimental::DataType::UNDEFINED);
}
#endif
