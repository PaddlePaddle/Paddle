#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/reduce_min_kernel.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/reduce.h"

namespace phi {

/**
 * @brief Binary min functor using min
 */
template <typename T>
struct CudaFastMinFunctor {
  inline T initial() { return static_cast<T>(std::numeric_limits<T>::max()); }

  __device__ __forceinline__ T operator()(const T a, const T b) const {
    return min(a, b);
  }
};

template <typename T, typename Context>
void MinRawKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const IntArray& dims,
                  bool keep_dim,
                  bool reduce_all,
                  DenseTensor* out) {
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  auto out_dtype = x.dtype();
  phi::Reduce<T, CudaFastMinFunctor, kps::IdentityFunctor>(
      dev_ctx, x, reduce_all, dims.GetData(), keep_dim, out_dtype, out);
}

}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(min_raw, KPS, ALL_LAYOUT, phi::MinRawKernel, float) {}
#else
PD_REGISTER_KERNEL(
    min_raw, KPS, ALL_LAYOUT, phi::MinRawKernel, float, double, int, int64_t) {}
#endif
