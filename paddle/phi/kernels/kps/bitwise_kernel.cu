/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/bitwise_kernel.h"

#ifdef PADDLE_WITH_XPU_KP
#include "paddle/phi/backends/xpu/xpu_context.h"
#else
#include "paddle/phi/backends/gpu/gpu_context.h"
#endif

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/bitwise_functors.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
namespace phi {

#define DEFINE_BITWISE_KERNEL(op_type)                      \
  template <typename T, typename Context>                   \
  void Bitwise##op_type##Kernel(const Context& dev_ctx,     \
                                const DenseTensor& x,       \
                                const DenseTensor& y,       \
                                DenseTensor* out) {         \
    dev_ctx.template Alloc<T>(out);                         \
    funcs::Bitwise##op_type##Functor<T> func;               \
    std::vector<const DenseTensor*> ins = {&x, &y};         \
    std::vector<DenseTensor*> outs = {out};                 \
    funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>( \
        dev_ctx, ins, &outs, -1, func);                     \
  }

DEFINE_BITWISE_KERNEL(And)
DEFINE_BITWISE_KERNEL(Or)
DEFINE_BITWISE_KERNEL(Xor)
#undef DEFINE_BITWISE_KERNEL

template <typename T, typename Context>
void BitwiseNotKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  funcs::BitwiseNotFunctor<T> unary_func;
  funcs::ElementwiseKernel<T, funcs::BitwiseNotFunctor<T>>(
      dev_ctx, ins, &outs, unary_func);
}

}  // namespace phi

#ifdef PADDLE_WITH_XPU_KP
PD_REGISTER_KERNEL(
    bitwise_and, KPS, ALL_LAYOUT, phi::BitwiseAndKernel, int, bool) {}
PD_REGISTER_KERNEL(
    bitwise_or, KPS, ALL_LAYOUT, phi::BitwiseOrKernel, int, bool) {}
PD_REGISTER_KERNEL(
    bitwise_xor, KPS, ALL_LAYOUT, phi::BitwiseXorKernel, int, bool) {}
PD_REGISTER_KERNEL(
    bitwise_not, KPS, ALL_LAYOUT, phi::BitwiseNotKernel, int, bool) {}

#else
PD_REGISTER_KERNEL(bitwise_and,
                   KPS,
                   ALL_LAYOUT,
                   phi::BitwiseAndKernel,
                   bool,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(bitwise_or,
                   KPS,
                   ALL_LAYOUT,
                   phi::BitwiseOrKernel,
                   bool,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(bitwise_xor,
                   KPS,
                   ALL_LAYOUT,
                   phi::BitwiseXorKernel,
                   bool,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

PD_REGISTER_KERNEL(bitwise_not,
                   KPS,
                   ALL_LAYOUT,
                   phi::BitwiseNotKernel,
                   bool,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}

#endif
