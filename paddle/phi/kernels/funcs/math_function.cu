#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <algorithm>
#include <vector>

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/math_function_impl.h"

namespace phi {
namespace funcs {

using DATATYPE = phi::dtype::float16;
#define blockM 16
#define blockN 16
#define accessSize 8

// __global__ void my_row_col_kernel1(
//     DATATYPE* output, const DATATYPE* input, int batch, int M, int N) {
//   __shared__ half aTile[blockM][blockN * 8 + 8];
//   int vol = M * N;
//   int batch_i = blockIdx.z;

//   // g_row_0 ,
//   //
//   g_col_0表示这个input矩阵的block的最左上角的全局的行号和列号，他们分别是16和16*8的倍数！！
//   int g_row_0 = blockIdx.x * blockDim.x;
//   int g_col_0 = blockIdx.y * blockDim.y * accessSize;

//   int thread_id_in_block = threadIdx.x + threadIdx.y * blockDim.x;
//   // local column and row read by the current cuda thread
//   int local_rowi = thread_id_in_block / 16;
//   int local_coli = thread_id_in_block % 16;

//   int g_row_i = g_row_0 + local_rowi;
//   int g_col_i = g_col_0 + local_coli * accessSize;

//   int input_offset = batch_i * vol + g_row_i * N + g_col_i;
//   if (g_row_i < M && g_col_i < N) {
//     *(float4*)(&aTile[local_rowi][local_coli * 8]) =
//         *(float4*)(input + input_offset);
//   }

//   __syncthreads();

//   // 将aTile_tmp看成这个矩阵blockM * (blockN * accessSize)
//   half* aTile_tmp = (half*)(aTile);
//   int a_row = blockM;
//   int a_col = blockN * accessSize + 8;

//   //   // 要变成col_i row_i 了
//   int new_idx = thread_id_in_block % 16;
//   int new_idy = thread_id_in_block / 16;
//   g_row_i = g_col_0 + new_idy;
//   g_col_i = g_row_0 + new_idx;

//   for (int i = 0; i < 8; i++) {
//     if (g_row_i >= N || g_col_i >= M) break;
//     int output_offset = batch_i * vol + g_row_i * M + g_col_i;
//     *(output + output_offset) = aTile_tmp[new_idx * a_col + new_idy];
//     g_row_i += 16;
//     new_idy += 16;
//   }
// }

// /*

// 思路是：将输入分成三个部分，M，N， batch这三个部分
// 每个thread block计算 blockM * (blockN * accessSize) 这么大部分！
// 每个thread block的线程维度是(blockM, blockN)
// 对于每个batch，其实就是将行矩阵存储变成列存储矩阵！
// */

// void my_row_col1(
//     DATATYPE* output, const DATATYPE* input, int batch, int M, int N) {
//   uint3 grid = {(M + blockM - 1) / blockM,
//                 (N + blockN * accessSize - 1) / (blockN * accessSize),
//                 batch};
//   uint3 block = {blockM, blockN, 1};
//   assert(N % 8 == 0);
//   my_row_col_kernel1<<<grid, block>>>(output, input, batch, M, N);
// }

// The following part of the code refers to NVIDIA-cutlass
// https://github.com/NVIDIA/cutlass/blob/master/tools/util/include/cutlass/util/device_nchw_to_nhwc.h
// Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights
// reserved. SPDX-License-Identifier: BSD-3-Clause
template <typename T>
__global__ void batch_transpose_kernel(
    T* output, const T* input, const int batch, const int M, const int N) {
  const int num = M * N;
  // "+1" to avoid smem bank conflict
  __shared__ T shbuf[32 * (32 + 1)];
  const int32_t tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int32_t wid = tid / 32;
  const int32_t lid = tid % 32;
  const int32_t batch_i = blockIdx.z;
  const int32_t mi0 = blockIdx.y * 32;
  const int32_t ni0 = blockIdx.x * 32;

  const size_t input_idx = batch_i * num + (mi0 + wid) * N + ni0;
  const T* A = input + input_idx;
  if (ni0 + lid < N) {
    const int lid_x_33 = lid * 33;
    if ((mi0 + 32) <= M) {
      int mi = wid;  // between 0 and 7
#pragma unroll
      for (int mLoopIdx = 0; mLoopIdx < 4; mLoopIdx++) {
        shbuf[lid_x_33 + mi] = A[lid];
        A = &A[8 * N];
        mi += 8;
      }
    } else {
      for (int mi = wid; mi < 32; mi += 8) {
        if ((mi + mi0) < M) {
          shbuf[lid_x_33 + mi] = A[lid];
        }
        A = &A[8 * N];
      }
    }
  }
  __syncthreads();

  const int32_t miOut = mi0 + lid;
  output = &output[batch_i * num + miOut];
  if (miOut < M) {
    if (ni0 + 32 < N) {
      int nI = wid;
#pragma unroll
      for (int nLoopIdx = 0; nLoopIdx < 4; ++nLoopIdx) {
        output[(ni0 + nI) * M] = shbuf[(nI)*33 + lid];
        nI += 8;
      }
    } else {
      for (int nI = wid; nI < 32; nI += 8) {
        if (ni0 + nI < N) {
          output[(ni0 + nI) * M] = shbuf[(nI)*33 + lid];
        }
      }
    }
  }
}

template <typename T>
void BatchTranspose(T* output, const T* input, int batch, int m, int n) {
  dim3 grid((n + 31) / 32, (m + 31) / 32, batch);
  dim3 block(32, 8);
  batch_transpose_kernel<<<grid, block>>>(output, input, batch, m, n);
}

using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;

template void BatchTranspose(
    float16* output, const float16* input, int batch, int m, int n);
template void BatchTranspose(
    float* output, const float* input, int batch, int m, int n);

template struct SetConstant<phi::GPUContext, float16>;
template struct SetConstant<phi::GPUContext, bfloat16>;
template struct SetConstant<phi::GPUContext, float>;
template struct SetConstant<phi::GPUContext, double>;
template struct SetConstant<phi::GPUContext, uint8_t>;
template struct SetConstant<phi::GPUContext, int>;
template struct SetConstant<phi::GPUContext, int16_t>;
template struct SetConstant<phi::GPUContext, int64_t>;
template struct SetConstant<phi::GPUContext, bool>;
template struct SetConstant<phi::GPUContext, phi::dtype::complex<float>>;
template struct SetConstant<phi::GPUContext, phi::dtype::complex<double>>;

template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, float16>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext,
                            bfloat16>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, float>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, double>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, uint8_t>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, int>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, int16_t>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, int64_t>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext, bool>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext,
                            phi::dtype::complex<float>>;
template struct SetConstant<paddle::platform::CUDAPinnedDeviceContext,
                            phi::dtype::complex<double>>;

#define DEFINE_GPU_TRANS(RANK)                                     \
  template struct Transpose<phi::GPUContext, bool, RANK>;          \
  template struct Transpose<phi::GPUContext, unsigned char, RANK>; \
  template struct Transpose<phi::GPUContext, float, RANK>;         \
  template struct Transpose<phi::GPUContext, double, RANK>;        \
  template struct Transpose<phi::GPUContext, float16, RANK>;       \
  template struct Transpose<phi::GPUContext, bfloat16, RANK>;      \
  template struct Transpose<phi::GPUContext, int8_t, RANK>;        \
  template struct Transpose<phi::GPUContext, int16_t, RANK>;       \
  template struct Transpose<phi::GPUContext, int32_t, RANK>;       \
  template struct Transpose<phi::GPUContext, int64_t, RANK>;       \
  template struct Transpose<phi::GPUContext,                       \
                            phi::dtype::complex<float>,            \
                            RANK>;                                 \
  template struct Transpose<phi::GPUContext, phi::dtype::complex<double>, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

#define REINTERPRET(T, DST_PTR, SRC_PTR) \
  T* DST_PTR = reinterpret_cast<T*>(SRC_PTR)

template <typename T>
__global__ void TransposeNormalKernel(const T* in_ptr,
                                      T* out_ptr,
                                      int64_t element,
                                      const int64_t* in_stride_ptr,
                                      const int64_t* out_stride_ptr,
                                      const int64_t* axis_ptr,
                                      int rank) {
  CUDA_KERNEL_LOOP(out_idx, element) {
    int64_t in_idx = 0;
    int64_t tmp_idx = out_idx;
    for (int i = 0; i < rank; ++i) {
      const int64_t coordinate = tmp_idx / out_stride_ptr[i];
      tmp_idx -= coordinate * out_stride_ptr[i];
      in_idx += coordinate * in_stride_ptr[axis_ptr[i]];
    }
    out_ptr[out_idx] = in_ptr[in_idx];
  }
}

template <typename DeviceContext, typename T>
void TransposeNormal<DeviceContext, T>::operator()(
    const DeviceContext& context,
    const phi::DenseTensor& in,
    phi::DenseTensor* out,
    const std::vector<int>& axis) {
  const int rank = axis.size();
  auto in_stride = phi::stride(in.dims());
  auto out_stride = phi::stride(out->dims());
  auto* in_ptr = in.data<T>();
  auto* out_ptr = out->data<T>();

  // copy in_stride, out_stride, axis to gpu device
  const paddle::platform::CUDAPlace& cuda_place = context.GetPlace();
  paddle::platform::CPUPlace cpu_place = paddle::platform::CPUPlace();
  size_t size = 3 * rank * sizeof(int64_t);
  auto cpu_buf_holder = paddle::memory::Alloc(cpu_place, size);
  auto cuda_buf_holder = paddle::memory::Alloc(cuda_place, size);
  REINTERPRET(int64_t, cpu_buf, cpu_buf_holder->ptr());
  REINTERPRET(int64_t, cuda_buf, cuda_buf_holder->ptr());
  for (int i = 0; i < rank; ++i) {
    cpu_buf[i] = in_stride[i];
    cpu_buf[rank + i] = out_stride[i];
    cpu_buf[2 * rank + i] = axis[i];
  }
  paddle::memory::Copy(
      cuda_place, cuda_buf, cpu_place, cpu_buf, size, context.stream());
  REINTERPRET(const int64_t, in_stride_ptr, cuda_buf);
  REINTERPRET(const int64_t, out_stride_ptr, cuda_buf + rank);
  REINTERPRET(const int64_t, axis_ptr, cuda_buf + 2 * rank);

  const int MAX_BLOCK_DIM = context.GetMaxThreadsPerBlock();
  const int MAX_GRID_DIM = context.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
  int64_t elements = in.numel();
  int block_size = (elements >= MAX_BLOCK_DIM)
                       ? MAX_BLOCK_DIM
                       : (1 << static_cast<int>(std::log2(elements)));
  int grid_size = elements / block_size;
  grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
  TransposeNormalKernel<T><<<grid_size, block_size, 0, context.stream()>>>(
      in_ptr, out_ptr, elements, in_stride_ptr, out_stride_ptr, axis_ptr, rank);
}

template <typename T>
struct TransposeNormal<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& in,
                  DenseTensor* out,
                  const std::vector<int>& axis) {
    const int rank = axis.size();
    auto in_stride = stride(in.dims());
    auto out_stride = stride(out->dims());
    auto* in_ptr = in.data<T>();
    auto* out_ptr = out->data<T>();

    // copy in_stride, out_stride, axis to gpu device
    const phi::GPUPlace& cuda_place = context.GetPlace();
    phi::CPUPlace cpu_place = paddle::platform::CPUPlace();
    size_t size = 3 * rank * sizeof(int64_t);
    auto cpu_buf_holder = paddle::memory::Alloc(cpu_place, size);
    auto cuda_buf_holder = paddle::memory::Alloc(cuda_place, size);
    REINTERPRET(int64_t, cpu_buf, cpu_buf_holder->ptr());
    REINTERPRET(int64_t, cuda_buf, cuda_buf_holder->ptr());
    for (int i = 0; i < rank; ++i) {
      cpu_buf[i] = in_stride[i];
      cpu_buf[rank + i] = out_stride[i];
      cpu_buf[2 * rank + i] = axis[i];
    }
    paddle::memory::Copy(
        cuda_place, cuda_buf, cpu_place, cpu_buf, size, context.stream());
    REINTERPRET(const int64_t, in_stride_ptr, cuda_buf);
    REINTERPRET(const int64_t, out_stride_ptr, cuda_buf + rank);
    REINTERPRET(const int64_t, axis_ptr, cuda_buf + 2 * rank);

    const int MAX_BLOCK_DIM = context.GetMaxThreadsPerBlock();
    const int MAX_GRID_DIM =
        context.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
    int64_t elements = in.numel();
    int block_size = (elements >= MAX_BLOCK_DIM)
                         ? MAX_BLOCK_DIM
                         : (1 << static_cast<int>(std::log2(elements)));
    int grid_size = elements / block_size;
    grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
    TransposeNormalKernel<T>
        <<<grid_size, block_size, 0, context.stream()>>>(in_ptr,
                                                         out_ptr,
                                                         elements,
                                                         in_stride_ptr,
                                                         out_stride_ptr,
                                                         axis_ptr,
                                                         rank);
  }
};

// define transpose normal
#define DEFINE_GPU_TRANS_NORMAL(TYPE) \
  template struct TransposeNormal<phi::GPUContext, TYPE>

DEFINE_GPU_TRANS_NORMAL(float16);
DEFINE_GPU_TRANS_NORMAL(bfloat16);
DEFINE_GPU_TRANS_NORMAL(float);
DEFINE_GPU_TRANS_NORMAL(double);
DEFINE_GPU_TRANS_NORMAL(int);
DEFINE_GPU_TRANS_NORMAL(int64_t);
DEFINE_GPU_TRANS_NORMAL(bool);
DEFINE_GPU_TRANS_NORMAL(int16_t);
DEFINE_GPU_TRANS_NORMAL(uint8_t);
DEFINE_GPU_TRANS_NORMAL(int8_t);
DEFINE_GPU_TRANS_NORMAL(phi::dtype::complex<float>);
DEFINE_GPU_TRANS_NORMAL(phi::dtype::complex<double>);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const paddle::platform::DeviceContext& context,
                       phi::DenseTensor* tensor,
                       float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void apply() const {
    SetConstant<phi::GPUContext, T> functor;
    functor(reinterpret_cast<const phi::GPUContext&>(context_),
            tensor_,
            static_cast<T>(value_));
  }

  const paddle::platform::DeviceContext& context_;
  phi::DenseTensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<paddle::platform::CUDAPlace>(
    const paddle::platform::DeviceContext& context,
    phi::DenseTensor* tensor,
    float value) {
  phi::VisitDataType(tensor->dtype(),
                     TensorSetConstantGPU(context, tensor, value));
}

template <typename T>
__global__ void RowwiseAddKernel(
    const T* a, const T* b, T* c, int width, int num) {
  T tmp = 1.0 / width;
  CUDA_KERNEL_LOOP(i, num) {
    int h = i * tmp;
    int w = i - h * width;
    c[i] = a[i] + b[w];
  }
}

template <typename T>
struct RowwiseAdd<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const phi::DenseTensor& input,
                  const phi::DenseTensor& vector,
                  phi::DenseTensor* output) {
    auto in_dims = input.dims();
    auto out_dims = output->dims();
    auto size = input.numel() / in_dims[0];
    PADDLE_ENFORCE_EQ(
        vector.numel(),
        size,
        phi::errors::InvalidArgument(
            "The input vector size"
            " should be equal to the size of each row of input tensor."
            " Expected vector size=%d, but received %d",
            size,
            vector.numel()));
    const char* in_dims_cstr = in_dims.to_str().c_str();
    const char* out_dims_cstr = out_dims.to_str().c_str();
    PADDLE_ENFORCE_EQ(
        out_dims,
        in_dims,
        phi::errors::InvalidArgument(
            "The output tensor shape should be same as the input tensor"
            " shape. Expected output tensor shape: %s,"
            " but received %s",
            in_dims_cstr,
            out_dims_cstr));
    int blocks = 512;
    int grids = (input.numel() + blocks - 1) / blocks;
    RowwiseAddKernel<T><<<grids, blocks, 0, context.stream()>>>(
        input.data<T>(),
        vector.data<T>(),
        output->data<T>(),
        static_cast<int>(in_dims[1]),
        static_cast<int>(input.numel()));
  }
};

template struct RowwiseAdd<phi::GPUContext, float>;
template struct RowwiseAdd<phi::GPUContext, double>;
template struct ColwiseSum<phi::GPUContext, float>;
template struct ColwiseSum<phi::GPUContext, int>;
template struct ColwiseSum<phi::GPUContext, int64_t>;
// template struct ColwiseSum<phi::GPUContext, double>;
// The ColwiseSum<phi::GPUContext, double> failed in debug
// mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<phi::GPUContext, double>::operator()(
    const phi::GPUContext& context,
    const phi::DenseTensor& input,
    phi::DenseTensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(),
                    size,
                    phi::errors::InvalidArgument(
                        "The size of input vector"
                        " should be equal to the size of input tensor column"
                        " dimension. Expected vector size=%d, but received %d",
                        size,
                        vector->numel()));
  phi::DenseTensor one;
  one.Resize({in_dims[0]});
  context.template Alloc<double>(&one);

  SetConstant<phi::GPUContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  phi::funcs::GetBlas<phi::GPUContext, double>(context).GEMV(
      true,
      static_cast<int>(in_dims[0]),
      static_cast<int>(in_dims[1]),
      1.0,
      input.data<double>(),
      one.data<double>(),
      0.0,
      vector->data<double>());
}

template struct RowwiseSum<phi::GPUContext, float>;
// template struct RowwiseSum<phi::GPUContext, double>;
// TODO(zcd): Following ColwiseSum format, need to confirm.
// The RowwiseSum<phi::GPUContext, double> failed in debug
// mode,
// and only failed for this case. So reimplemented it.
template <>
void RowwiseSum<phi::GPUContext, double>::operator()(
    const phi::GPUContext& context,
    const phi::DenseTensor& input,
    phi::DenseTensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(),
                    in_dims[0],
                    phi::errors::InvalidArgument(
                        "The size of input vector"
                        " should be equal to the size of input tensor row"
                        " dimension. Expected vector size=%d, but received %d",
                        in_dims[0],
                        vector->numel()));
  phi::DenseTensor one;
  one.Resize({size});
  context.template Alloc<double>(&one);

  SetConstant<phi::GPUContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  phi::funcs::GetBlas<phi::GPUContext, double>(context).GEMV(
      true,
      static_cast<int>(in_dims[1]),
      static_cast<int>(in_dims[0]),
      1.0,
      one.data<double>(),
      input.data<double>(),
      0.0,
      vector->data<double>());
}

template struct RowwiseMean<phi::GPUContext, float>;
template struct RowwiseMean<phi::GPUContext, double>;

}  // namespace funcs
}  // namespace phi
