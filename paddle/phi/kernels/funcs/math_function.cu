#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/phi/kernels/funcs/math_function.h"
#include <float.h>
#include <algorithm>
#include <vector>
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/memory_utils.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/math_function_impl.h"

namespace phi {
namespace funcs {

// The following part of the code refers to NVIDIA-cutlass
// https://github.com/NVIDIA/cutlass/blob/master/tools/util/include/cutlass/util/device_nchw_to_nhwc.h
// Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights
// reserved. SPDX-License-Identifier: BSD-3-Clause
template <typename T>
__global__ void batch_transpose_kernel(T* output,
                                       const T* input,
                                       const int batch,
                                       const int M,
                                       const int N,
                                       int swizzle) {
  const int num = M * N;
  // "+1" to avoid smem bank conflict
  __shared__ T shbuf[32 * (32 + 1)];
  const int32_t tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int32_t wid = tid / 32;
  const int32_t lid = tid % 32;
  const int32_t batch_i = blockIdx.z;
  const int32_t mi0 = (blockIdx.y * swizzle + blockIdx.x % swizzle) * 32;
  const int32_t ni0 = blockIdx.x / swizzle * 32;

  const size_t input_idx = batch_i * num + (mi0 + wid) * N + ni0;
  const T* A = input + input_idx;
  if (ni0 + lid < N) {
    const int lid_x_33 = lid * 33;
    if ((mi0 + 32) <= M) {
      int mi = wid;  // between 0 and 7
#pragma unroll
      for (int mLoopIdx = 0; mLoopIdx < 4; mLoopIdx++) {
        shbuf[lid_x_33 + mi] = A[lid];
        A = &A[8 * N];
        mi += 8;
      }
    } else {
      for (int mi = wid; mi < 32; mi += 8) {
        if ((mi + mi0) < M) {
          shbuf[lid_x_33 + mi] = A[lid];
        }
        A = &A[8 * N];
      }
    }
  }
  __syncthreads();

  const int32_t miOut = mi0 + lid;
  output = &output[batch_i * num + miOut];
  if (miOut < M) {
    if (ni0 + 32 < N) {
      int nI = wid;
#pragma unroll
      for (int nLoopIdx = 0; nLoopIdx < 4; ++nLoopIdx) {
        output[(ni0 + nI) * M] = shbuf[(nI)*33 + lid];
        nI += 8;
      }
    } else {
      for (int nI = wid; nI < 32; nI += 8) {
        if (ni0 + nI < N) {
          output[(ni0 + nI) * M] = shbuf[(nI)*33 + lid];
        }
      }
    }
  }
}

template <typename T>
void BatchTranspose(T* output,
                    const T* input,
                    int64_t batch,
                    int64_t m,
                    int64_t n,
                    const phi::GPUContext* dev_ctx) {
  int64_t device_id = dev_ctx->GetPlace().GetDeviceId();
  const auto& prop = phi::backends::gpu::GetDeviceProperties(device_id);
  int max_grid_y = prop.maxGridSize[1];
  int64_t input_num = batch * m * n;

  if (input_num >= std::numeric_limits<int>::max()) {
    PADDLE_THROW(phi::errors::Unimplemented(
        "Unsupported input size, batch: %ld,m: %ld, n: %ld", batch, m, n));
  }

  dim3 logical_grid((n + 31) / 32, (m + 31) / 32, batch);
  dim3 block(32, 8);
  // we set swizzle to 2 default.
  int swizzle = (logical_grid.y + max_grid_y - 1) / max_grid_y;
  swizzle = std::max(swizzle, 2);
  dim3 physical_grid(logical_grid.x * swizzle,
                     (logical_grid.y + swizzle - 1) / swizzle,
                     batch);
  batch_transpose_kernel<<<physical_grid, block>>>(
      output, input, batch, m, n, swizzle);
}

using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;

template void BatchTranspose(float16* output,
                             const float16* input,
                             int64_t batch,
                             int64_t m,
                             int64_t n,
                             const phi::GPUContext* dev_ctx);
template void BatchTranspose(float* output,
                             const float* input,
                             int64_t batch,
                             int64_t m,
                             int64_t n,
                             const phi::GPUContext* dev_ctx);
template void BatchTranspose(bfloat16* output,
                             const bfloat16* input,
                             int64_t batch,
                             int64_t m,
                             int64_t n,
                             const phi::GPUContext* dev_ctx);

template struct SetConstant<phi::GPUContext, float16>;
template struct SetConstant<phi::GPUContext, bfloat16>;
template struct SetConstant<phi::GPUContext, float>;
template struct SetConstant<phi::GPUContext, double>;
template struct SetConstant<phi::GPUContext, uint8_t>;
template struct SetConstant<phi::GPUContext, int8_t>;
template struct SetConstant<phi::GPUContext, int>;
template struct SetConstant<phi::GPUContext, int16_t>;
template struct SetConstant<phi::GPUContext, int64_t>;
template struct SetConstant<phi::GPUContext, bool>;
template struct SetConstant<phi::GPUContext, phi::dtype::complex<float>>;
template struct SetConstant<phi::GPUContext, phi::dtype::complex<double>>;

template struct SetConstant<phi::GPUPinnedContext, float16>;
template struct SetConstant<phi::GPUPinnedContext, bfloat16>;
template struct SetConstant<phi::GPUPinnedContext, float>;
template struct SetConstant<phi::GPUPinnedContext, double>;
template struct SetConstant<phi::GPUPinnedContext, uint8_t>;
template struct SetConstant<phi::GPUPinnedContext, int8_t>;
template struct SetConstant<phi::GPUPinnedContext, int>;
template struct SetConstant<phi::GPUPinnedContext, int16_t>;
template struct SetConstant<phi::GPUPinnedContext, int64_t>;
template struct SetConstant<phi::GPUPinnedContext, bool>;
template struct SetConstant<phi::GPUPinnedContext, phi::dtype::complex<float>>;
template struct SetConstant<phi::GPUPinnedContext, phi::dtype::complex<double>>;

#define DEFINE_GPU_TRANS(RANK)                                     \
  template struct Transpose<phi::GPUContext, bool, RANK>;          \
  template struct Transpose<phi::GPUContext, unsigned char, RANK>; \
  template struct Transpose<phi::GPUContext, float, RANK>;         \
  template struct Transpose<phi::GPUContext, double, RANK>;        \
  template struct Transpose<phi::GPUContext, float16, RANK>;       \
  template struct Transpose<phi::GPUContext, bfloat16, RANK>;      \
  template struct Transpose<phi::GPUContext, int8_t, RANK>;        \
  template struct Transpose<phi::GPUContext, int16_t, RANK>;       \
  template struct Transpose<phi::GPUContext, int32_t, RANK>;       \
  template struct Transpose<phi::GPUContext, int64_t, RANK>;       \
  template struct Transpose<phi::GPUContext,                       \
                            phi::dtype::complex<float>,            \
                            RANK>;                                 \
  template struct Transpose<phi::GPUContext, phi::dtype::complex<double>, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

#define REINTERPRET(T, DST_PTR, SRC_PTR) \
  T* DST_PTR = reinterpret_cast<T*>(SRC_PTR)

template <typename T>
__global__ void TransposeNormalKernel(const T* in_ptr,
                                      T* out_ptr,
                                      int64_t element,
                                      const int64_t* in_stride_ptr,
                                      const int64_t* out_stride_ptr,
                                      const int64_t* axis_ptr,
                                      int rank) {
  CUDA_KERNEL_LOOP(out_idx, element) {
    int64_t in_idx = 0;
    int64_t tmp_idx = out_idx;
    for (int i = 0; i < rank; ++i) {
      const int64_t coordinate = tmp_idx / out_stride_ptr[i];
      tmp_idx -= coordinate * out_stride_ptr[i];
      in_idx += coordinate * in_stride_ptr[axis_ptr[i]];
    }
    out_ptr[out_idx] = in_ptr[in_idx];
  }
}

template <typename DeviceContext, typename T>
void TransposeNormal<DeviceContext, T>::operator()(
    const DeviceContext& context,
    const phi::DenseTensor& in,
    phi::DenseTensor* out,
    const std::vector<int>& axis) {
  const int rank = axis.size();
  auto in_stride = phi::stride(in.dims());
  auto out_stride = phi::stride(out->dims());
  auto* in_ptr = in.data<T>();
  auto* out_ptr = out->data<T>();

  // copy in_stride, out_stride, axis to gpu device
  const phi::GPUPlace& cuda_place = context.GetPlace();
  phi::CPUPlace cpu_place = phi::CPUPlace();
  size_t size = 3 * rank * sizeof(int64_t);
  auto cpu_buf_holder = phi::memory_utils::Alloc(cpu_place, size);
  auto cuda_buf_holder = phi::memory_utils::Alloc(cuda_place, size);
  REINTERPRET(int64_t, cpu_buf, cpu_buf_holder->ptr());
  REINTERPRET(int64_t, cuda_buf, cuda_buf_holder->ptr());
  for (int i = 0; i < rank; ++i) {
    cpu_buf[i] = in_stride[i];
    cpu_buf[rank + i] = out_stride[i];
    cpu_buf[2 * rank + i] = axis[i];
  }
  memory_utils::Copy(
      cuda_place, cuda_buf, cpu_place, cpu_buf, size, context.stream());
  REINTERPRET(const int64_t, in_stride_ptr, cuda_buf);
  REINTERPRET(const int64_t, out_stride_ptr, cuda_buf + rank);
  REINTERPRET(const int64_t, axis_ptr, cuda_buf + 2 * rank);

  const int MAX_BLOCK_DIM = context.GetMaxThreadsPerBlock();
  const int MAX_GRID_DIM = context.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
  int64_t elements = in.numel();
  int block_size = (elements >= MAX_BLOCK_DIM)
                       ? MAX_BLOCK_DIM
                       : (1 << static_cast<int>(std::log2(elements)));
  int grid_size = elements / block_size;
  grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
  TransposeNormalKernel<T><<<grid_size, block_size, 0, context.stream()>>>(
      in_ptr, out_ptr, elements, in_stride_ptr, out_stride_ptr, axis_ptr, rank);
}

template <typename T>
struct TransposeNormal<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const DenseTensor& in,
                  DenseTensor* out,
                  const std::vector<int>& axis) {
    const int rank = axis.size();
    auto in_stride = stride(in.dims());
    auto out_stride = stride(out->dims());
    auto* in_ptr = in.data<T>();
    auto* out_ptr = out->data<T>();

    // copy in_stride, out_stride, axis to gpu device
    const phi::GPUPlace& cuda_place = context.GetPlace();
    phi::CPUPlace cpu_place = phi::CPUPlace();
    size_t size = 3 * rank * sizeof(int64_t);
    auto cpu_buf_holder = phi::memory_utils::Alloc(cpu_place, size);
    auto cuda_buf_holder = phi::memory_utils::Alloc(cuda_place, size);
    REINTERPRET(int64_t, cpu_buf, cpu_buf_holder->ptr());
    REINTERPRET(int64_t, cuda_buf, cuda_buf_holder->ptr());
    for (int i = 0; i < rank; ++i) {
      cpu_buf[i] = in_stride[i];
      cpu_buf[rank + i] = out_stride[i];
      cpu_buf[2 * rank + i] = axis[i];
    }
    memory_utils::Copy(
        cuda_place, cuda_buf, cpu_place, cpu_buf, size, context.stream());
    REINTERPRET(const int64_t, in_stride_ptr, cuda_buf);
    REINTERPRET(const int64_t, out_stride_ptr, cuda_buf + rank);
    REINTERPRET(const int64_t, axis_ptr, cuda_buf + 2 * rank);

    const int MAX_BLOCK_DIM = context.GetMaxThreadsPerBlock();
    const int MAX_GRID_DIM =
        context.GetMaxPhysicalThreadCount() / MAX_BLOCK_DIM;
    int64_t elements = in.numel();
    int block_size = (elements >= MAX_BLOCK_DIM)
                         ? MAX_BLOCK_DIM
                         : (1 << static_cast<int>(std::log2(elements)));
    int grid_size = elements / block_size;
    grid_size = (grid_size >= MAX_GRID_DIM) ? MAX_GRID_DIM : grid_size;
    TransposeNormalKernel<T>
        <<<grid_size, block_size, 0, context.stream()>>>(in_ptr,
                                                         out_ptr,
                                                         elements,
                                                         in_stride_ptr,
                                                         out_stride_ptr,
                                                         axis_ptr,
                                                         rank);
  }
};

// define transpose normal
#define DEFINE_GPU_TRANS_NORMAL(TYPE) \
  template struct TransposeNormal<phi::GPUContext, TYPE>

DEFINE_GPU_TRANS_NORMAL(float16);
DEFINE_GPU_TRANS_NORMAL(bfloat16);
DEFINE_GPU_TRANS_NORMAL(float);
DEFINE_GPU_TRANS_NORMAL(double);
DEFINE_GPU_TRANS_NORMAL(int);
DEFINE_GPU_TRANS_NORMAL(int64_t);
DEFINE_GPU_TRANS_NORMAL(bool);
DEFINE_GPU_TRANS_NORMAL(int16_t);
DEFINE_GPU_TRANS_NORMAL(uint8_t);
DEFINE_GPU_TRANS_NORMAL(int8_t);
DEFINE_GPU_TRANS_NORMAL(phi::dtype::complex<float>);
DEFINE_GPU_TRANS_NORMAL(phi::dtype::complex<double>);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const phi::DeviceContext& context,
                       phi::DenseTensor* tensor,
                       float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void apply() const {
    SetConstant<phi::GPUContext, T> functor;
    functor(reinterpret_cast<const phi::GPUContext&>(context_),
            tensor_,
            static_cast<T>(value_));
  }

  const phi::DeviceContext& context_;
  phi::DenseTensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<phi::GPUPlace>(const phi::DeviceContext& context,
                                            phi::DenseTensor* tensor,
                                            float value) {
  phi::VisitDataType(tensor->dtype(),
                     TensorSetConstantGPU(context, tensor, value));
}

template <typename T>
__global__ void RowwiseAddKernel(
    const T* a, const T* b, T* c, int width, int num) {
  T tmp = 1.0 / width;
  CUDA_KERNEL_LOOP(i, num) {
    int h = i * tmp;
    int w = i - h * width;
    c[i] = a[i] + b[w];
  }
}

template <typename T>
struct RowwiseAdd<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const phi::DenseTensor& input,
                  const phi::DenseTensor& vector,
                  phi::DenseTensor* output) {
    auto in_dims = input.dims();
    auto out_dims = output->dims();
    auto size = input.numel() / in_dims[0];
    PADDLE_ENFORCE_EQ(
        vector.numel(),
        size,
        phi::errors::InvalidArgument(
            "The input vector size"
            " should be equal to the size of each row of input tensor."
            " Expected vector size=%d, but received %d",
            size,
            vector.numel()));
    const char* in_dims_cstr = in_dims.to_str().c_str();
    const char* out_dims_cstr = out_dims.to_str().c_str();
    PADDLE_ENFORCE_EQ(
        out_dims,
        in_dims,
        phi::errors::InvalidArgument(
            "The output tensor shape should be same as the input tensor"
            " shape. Expected output tensor shape: %s,"
            " but received %s",
            in_dims_cstr,
            out_dims_cstr));
    int blocks = 512;
    int grids = (input.numel() + blocks - 1) / blocks;
    RowwiseAddKernel<T><<<grids, blocks, 0, context.stream()>>>(
        input.data<T>(),
        vector.data<T>(),
        output->data<T>(),
        static_cast<int>(in_dims[1]),
        static_cast<int>(input.numel()));
  }
};

template struct RowwiseAdd<phi::GPUContext, float>;
template struct RowwiseAdd<phi::GPUContext, double>;
template struct ColwiseSum<phi::GPUContext, float>;
template struct ColwiseSum<phi::GPUContext, int>;
template struct ColwiseSum<phi::GPUContext, int64_t>;
// template struct ColwiseSum<phi::GPUContext, double>;
// The ColwiseSum<phi::GPUContext, double> failed in debug
// mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<phi::GPUContext, double>::operator()(
    const phi::GPUContext& context,
    const phi::DenseTensor& input,
    phi::DenseTensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(),
                    size,
                    phi::errors::InvalidArgument(
                        "The size of input vector"
                        " should be equal to the size of input tensor column"
                        " dimension. Expected vector size=%d, but received %d",
                        size,
                        vector->numel()));
  phi::DenseTensor one;
  one.Resize({in_dims[0]});
  context.template Alloc<double>(&one);

  SetConstant<phi::GPUContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  phi::funcs::GetBlas<phi::GPUContext, double>(context).GEMV(
      true,
      static_cast<int>(in_dims[0]),
      static_cast<int>(in_dims[1]),
      1.0,
      input.data<double>(),
      one.data<double>(),
      0.0,
      vector->data<double>());
}

template struct RowwiseSum<phi::GPUContext, float>;
// template struct RowwiseSum<phi::GPUContext, double>;
// TODO(zcd): Following ColwiseSum format, need to confirm.
// The RowwiseSum<phi::GPUContext, double> failed in debug
// mode,
// and only failed for this case. So reimplemented it.
template <>
void RowwiseSum<phi::GPUContext, double>::operator()(
    const phi::GPUContext& context,
    const phi::DenseTensor& input,
    phi::DenseTensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(),
                    in_dims[0],
                    phi::errors::InvalidArgument(
                        "The size of input vector"
                        " should be equal to the size of input tensor row"
                        " dimension. Expected vector size=%d, but received %d",
                        in_dims[0],
                        vector->numel()));
  phi::DenseTensor one;
  one.Resize({size});
  context.template Alloc<double>(&one);

  SetConstant<phi::GPUContext, double> set;
  set(context, &one, static_cast<double>(1.0));
  phi::funcs::GetBlas<phi::GPUContext, double>(context).GEMV(
      true,
      static_cast<int>(in_dims[1]),
      static_cast<int>(in_dims[0]),
      1.0,
      one.data<double>(),
      input.data<double>(),
      0.0,
      vector->data<double>());
}

template struct RowwiseMean<phi::GPUContext, float>;
template struct RowwiseMean<phi::GPUContext, double>;

}  // namespace funcs
}  // namespace phi
