#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#ifndef _USE_MATH_DEFINES
#define _USE_MATH_DEFINES
#endif
#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/funcs/eigen/eigen_function.h"
#include "paddle/phi/kernels/funcs/eigen/extensions.h"

namespace phi {
namespace funcs {

template <typename T>
struct EigenErf<Eigen::GpuDevice, T> {
  using InType = Eigen::TensorMap<
      Eigen::Tensor<const T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  using OutType =
      Eigen::TensorMap<Eigen::Tensor<T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  static void Eval(const Eigen::GpuDevice& dev, OutType out, const InType& in) {
    out.device(dev) = in.erf();
  }
};

template <typename T>
struct EigenErfGrad<Eigen::GpuDevice, T> {
  using InType = Eigen::TensorMap<
      Eigen::Tensor<const T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  using OutType =
      Eigen::TensorMap<Eigen::Tensor<T, 1, Eigen::RowMajor, Eigen::DenseIndex>>;
  static void Eval(const Eigen::GpuDevice& dev,
                   OutType din,
                   const InType& in,
                   const InType& dout) {
    din.device(dev) =
        dout * static_cast<T>(M_2_SQRTPI) * (-(in.square())).exp();
  }
};

#define INSTANTIATION(FUNCTOR)                       \
  template struct FUNCTOR<Eigen::GpuDevice, float>;  \
  template struct FUNCTOR<Eigen::GpuDevice, double>; \
  template struct FUNCTOR<Eigen::GpuDevice, dtype::float16>
INSTANTIATION(EigenErf);
INSTANTIATION(EigenErfGrad);
#undef INSTANTIATION

}  // namespace funcs
}  // namespace phi
