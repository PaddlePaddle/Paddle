#include "hip/hip_runtime.h"
/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "phi/kernels/funcs/skip_layernorm_functor.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/device_context.h"

namespace phi {
namespace funcs {

template <typename T>
void SkipLayerNormFunctor<T>::operator()(const int num,
                                         const int hidden,
                                         const T *input1,
                                         const T *input2,
                                         const T *scale,
                                         const T *bias,
                                         T *output,
                                         float eps,
                                         gpuStream_t stream) {
  int block = num / hidden;
  if (hidden <= WARP_SIZE) {
    const int threads = WARP_SIZE;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else if (hidden <= 128) {
    const int threads = 128;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else if (hidden == 384) {
    const int threads = 384;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else {
    const int threads = 256;
    if (hidden % 2 == 0) {
      if (std::is_same<T, float>::value) {
        SkipLayerNormKernel2<float, float2, threads>
            <<<block, threads, 0, stream>>>(
                num,
                hidden / 2,
                reinterpret_cast<const float2 *>(input1),
                reinterpret_cast<const float2 *>(input2),
                reinterpret_cast<float2 *>(output),
                reinterpret_cast<const float2 *>(scale),
                reinterpret_cast<const float2 *>(bias),
                eps);
// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#ifndef __HIPCC__
      } else if (std::is_same<T, __half>::value) {
        SkipLayerNormKernel2<__half, __half2, threads>
            <<<block, threads, 0, stream>>>(
                num,
                hidden / 2,
                reinterpret_cast<const __half2 *>(input1),
                reinterpret_cast<const __half2 *>(input2),
                reinterpret_cast<__half2 *>(output),
                reinterpret_cast<const __half2 *>(scale),
                reinterpret_cast<const __half2 *>(bias),
                eps);
#endif
      } else {
        assert(false);
        // should not be here
      }
    } else {
      SkipLayerNormKernel<T, threads><<<block, threads, 0, stream>>>(
          num, hidden, input1, input2, output, scale, bias, eps);
    }
  }
}

template class SkipLayerNormFunctor<float>;

// device function 'operator()' is not supportted until cuda 10.0
// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 10000
template class SkipLayerNormFunctor<half>;
#endif

}  // namespace funcs
}  // namespace phi
