#include "hip/hip_runtime.h"
/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/funcs/skip_layernorm_functor.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
namespace phi {
namespace funcs {

template <typename T>
__device__ __forceinline__ T local_rsqrt(T num) {
  return rsqrt(static_cast<float>(num));
}
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
__device__ __forceinline__ half local_rsqrt(half num) { return hrsqrt(num); }
#endif

template <typename T, int TPB>
__device__ inline void LayerNorm(const phi::funcs::kvp<T> &thread_data,
                                 const int ld,
                                 const int offset,
                                 const T *bias,
                                 const T *scale,
                                 T *output,
                                 T eps) {
  using BlockReduce = hipcub::BlockReduce<phi::funcs::kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = local_rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    const T val = output[idx];
    const T g(scale[i]);
    const T b(bias[i]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, unsigned TPB>
__global__ void SkipLayerNormKernel(int num,
                                    int hidden,
                                    const T *input1,
                                    const T *input2,
                                    T *output,
                                    const T *scale,
                                    const T *bias,
                                    T eps) {
  const T rld = T(1) / T(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<T> thread_data(0, 0);

  for (int it = threadIdx.x; it < hidden; it += TPB) {
    const int idx = offset + it;
    const T val = input1[idx] + input2[idx];
    const T rldval = rld * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<T>(rldval, rldval * val));
    output[idx] = val;
  }
  LayerNorm<T, TPB>(thread_data, hidden, offset, bias, scale, output, eps);
}

// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#ifndef __HIPCC__  // @{ Half kernel: SkipLayerNormKernel
template <>
__global__ void SkipLayerNormKernel<half, 256>(int num,
                                               int hidden,
                                               const half *input1,
                                               const half *input2,
                                               half *output,
                                               const half *scale,
                                               const half *bias,
                                               half eps) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const half rld = half(1) / half(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<half> thread_data(0, 0);

  for (int it = threadIdx.x; it < hidden; it += 256) {
    const int idx = offset + it;
    const half val = input1[idx] + input2[idx];
    const half rldval = rld * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<half>(rldval, rldval * val));
    output[idx] = val;
  }
  LayerNorm<half, 256>(thread_data, hidden, offset, bias, scale, output, eps);
#endif
}
#endif  // @} End Half kernel: SkipLayerNormKernel

template <typename T, typename T2, int TPB>
__device__ inline void LayerNorm2(const phi::funcs::kvp<T> &thread_data,
                                  const int ld,
                                  const int offset,
                                  const T2 *bias,
                                  const T2 *scale,
                                  T2 *output,
                                  T eps) {
  using BlockReduce = hipcub::BlockReduce<phi::funcs::kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = local_rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    T2 val = output[idx];
    const T2 g = scale[i];
    const T2 b = bias[i];
    val.x = T(g.x) * (val.x - mu) * rsigma + T(b.x);
    val.y = T(g.y) * (val.y - mu) * rsigma + T(b.y);
    output[idx] = val;
  }
}

template <typename T, typename T2, unsigned TPB>
__global__ void SkipLayerNormKernel2(int num,
                                     int hidden,
                                     const T2 *input1,
                                     const T2 *input2,
                                     T2 *output,
                                     const T2 *scale,
                                     const T2 *bias,
                                     float eps) {
  const T rld = T(0.5f / hidden);  // because hidden is hidden/2
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<T> thread_data(0, 0);

  for (int it = threadIdx.x; it < hidden; it += TPB) {
    const int idx = offset + it;
    const T2 val2 = input1[idx] + input2[idx];
    thread_data = pair_sum(
        thread_data,
        phi::funcs::kvp<T>(rld * (val2.x + val2.y),
                           rld * val2.x * val2.x + rld * val2.y * val2.y));
    output[idx] = val2;
  }
  LayerNorm2<T, T2, TPB>(thread_data, hidden, offset, bias, scale, output, eps);
}

// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#ifndef __HIPCC__  // @{ Half kernel: SkipLayerNormKernel2
template <>
__global__ void SkipLayerNormKernel2<half, half2, 256>(int num,
                                                       int hidden,
                                                       const half2 *input1,
                                                       const half2 *input2,
                                                       half2 *output,
                                                       const half2 *scale,
                                                       const half2 *bias,
                                                       float eps) {
// operator "+" of half only suppotted after cuda version 10.0
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__) && CUDA_VERSION >= 10000
  const half rld = half(0.5f / hidden);  // because hidden is hidden/2
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<half> thread_data(0, 0);

  for (int it = threadIdx.x; it < hidden; it += 256) {
    const int idx = offset + it;
    const half2 val2 = input1[idx] + input2[idx];
    thread_data = pair_sum(
        thread_data,
        phi::funcs::kvp<half>(rld * (val2.x + val2.y),
                              rld * val2.x * val2.x + rld * val2.y * val2.y));
    output[idx] = val2;
  }
  LayerNorm2<half, half2, 256>(
      thread_data, hidden, offset, bias, scale, output, eps);
#endif
}
#endif  // @} End Half kernel: SkipLayerNormKernel2

template <typename T, int TPB>
__device__ inline void LayerNormSmall(T val,
                                      const phi::funcs::kvp<T> &thread_data,
                                      const int ld,
                                      const int idx,
                                      const T *bias,
                                      const T *scale,
                                      T *output,
                                      T eps) {
  using BlockReduce = hipcub::BlockReduce<phi::funcs::kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = local_rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  if (threadIdx.x < ld) {
    const T g(scale[threadIdx.x]);
    const T b(bias[threadIdx.x]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, unsigned TPB>
__global__ void SkipLayerNormSmallKernel(int num,
                                         int hidden,
                                         const T *input1,
                                         const T *input2,
                                         T *output,
                                         const T *scale,
                                         const T *bias,
                                         T eps) {
  const T rld = T(1) / T(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<T> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  T val = 0;
  if (threadIdx.x < hidden) {
    val = input1[idx] + input2[idx];
    const T rldval = rld * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<T>(rldval, rldval * val));
  }
  LayerNormSmall<T, TPB>(
      val, thread_data, hidden, idx, bias, scale, output, eps);
}

// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#ifndef __HIPCC__  // @{ Half kernel: SkipLayerNormSmallKernel
template <>
__global__ void SkipLayerNormSmallKernel<half, 32>(int num,
                                                   int hidden,
                                                   const half *input1,
                                                   const half *input2,
                                                   half *output,
                                                   const half *scale,
                                                   const half *bias,
                                                   half eps) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const half rld = half(1) / half(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<half> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  half val = 0;
  if (threadIdx.x < hidden) {
    val = input1[idx] + input2[idx];
    const half rldval = rld * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<half>(rldval, rldval * val));
  }
  LayerNormSmall<half, 32>(
      val, thread_data, hidden, idx, bias, scale, output, eps);
#endif
}

template <>
__global__ void SkipLayerNormSmallKernel<half, 128>(int num,
                                                    int hidden,
                                                    const half *input1,
                                                    const half *input2,
                                                    half *output,
                                                    const half *scale,
                                                    const half *bias,
                                                    half eps) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const half rld = half(1) / half(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<half> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  half val = 0;
  if (threadIdx.x < hidden) {
    val = input1[idx] + input2[idx];
    const half rldval = rld * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<half>(rldval, rldval * val));
  }
  LayerNormSmall<half, 128>(
      val, thread_data, hidden, idx, bias, scale, output, eps);
#endif
}

template <>
__global__ void SkipLayerNormSmallKernel<half, 384>(int num,
                                                    int hidden,
                                                    const half *input1,
                                                    const half *input2,
                                                    half *output,
                                                    const half *scale,
                                                    const half *bias,
                                                    half eps) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const half rld = half(1) / half(hidden);
  const int offset = blockIdx.x * hidden;
  hipcub::Sum pair_sum;
  phi::funcs::kvp<half> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  half val = 0;
  if (threadIdx.x < hidden) {
    val = input1[idx] + input2[idx];
    const half rldval = rld * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<half>(rldval, rldval * val));
  }
  LayerNormSmall<half, 384>(
      val, thread_data, hidden, idx, bias, scale, output, eps);
#endif
}
#endif  // @} End Half kernel: SkipLayerNormSmallKernel

template <typename T>
void SkipLayerNormFunctor<T>::operator()(const int num,
                                         const int hidden,
                                         const T *input1,
                                         const T *input2,
                                         const T *scale,
                                         const T *bias,
                                         T *output,
                                         float eps,
                                         gpuStream_t stream) {
  int block = num / hidden;
  if (hidden <= WARP_SIZE) {
    const int threads = WARP_SIZE;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else if (hidden <= 128) {
    const int threads = 128;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else if (hidden == 384) {
    const int threads = 384;
    SkipLayerNormSmallKernel<T, threads><<<block, threads, 0, stream>>>(
        num, hidden, input1, input2, output, scale, bias, eps);
  } else {
    const int threads = 256;
    if (hidden % 2 == 0) {
      if (std::is_same<T, float>::value) {
        SkipLayerNormKernel2<float, float2, threads>
            <<<block, threads, 0, stream>>>(
                num,
                hidden / 2,
                reinterpret_cast<const float2 *>(input1),
                reinterpret_cast<const float2 *>(input2),
                reinterpret_cast<float2 *>(output),
                reinterpret_cast<const float2 *>(scale),
                reinterpret_cast<const float2 *>(bias),
                eps);
// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#ifndef __HIPCC__
      } else if (std::is_same<T, __half>::value) {
        SkipLayerNormKernel2<__half, __half2, threads>
            <<<block, threads, 0, stream>>>(
                num,
                hidden / 2,
                reinterpret_cast<const __half2 *>(input1),
                reinterpret_cast<const __half2 *>(input2),
                reinterpret_cast<__half2 *>(output),
                reinterpret_cast<const __half2 *>(scale),
                reinterpret_cast<const __half2 *>(bias),
                eps);
#endif
      } else {
        assert(false);
        // should not be here
      }
    } else {
      SkipLayerNormKernel<T, threads><<<block, threads, 0, stream>>>(
          num, hidden, input1, input2, output, scale, bias, eps);
    }
  }
}

template class SkipLayerNormFunctor<float>;

// device function 'operator()' is not supportted until cuda 10.0
// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 10000
template class SkipLayerNormFunctor<half>;
#endif

}  // namespace funcs
}  // namespace phi
