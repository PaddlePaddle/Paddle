#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/funcs/concat_and_split_functor.h"

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/platform/cuda_graph_with_memory_pool.h"

namespace phi {
namespace funcs {

template <typename T>
__global__ void ConcatKernel_(const T** inputs,
                              const int64_t* input_cols,
                              int col_size,
                              const int64_t output_rows,
                              const int64_t output_cols,
                              T* output) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int curr_segment = 0;
  int curr_offset = input_cols[0];
  for (; tid_x < output_cols; tid_x += blockDim.x * gridDim.x) {
    int curr_col_offset = input_cols[curr_segment + 1];
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = input_cols[curr_segment + 1];
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;

    const T* input_ptr = inputs[curr_segment];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < output_rows; tid_y += blockDim.y * gridDim.y)
      output[tid_y * output_cols + tid_x] =
          input_ptr[tid_y * segment_width + local_col];
  }
}

template <typename T>
__device__ void ConcatKernelDetail(const T** inputs_data,
                                   const int fixed_in_col,
                                   const int out_rows,
                                   const int out_cols,
                                   T* output_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  for (; tid_x < out_cols; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x * 1.0 / fixed_in_col;
    int in_offset = tid_x - split * fixed_in_col;
    const T* input_ptr = inputs_data[split];
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < out_rows; tid_y += blockDim.y * gridDim.y) {
      output_data[tid_y * out_cols + tid_x] =
          input_ptr[tid_y * fixed_in_col + in_offset];
    }
  }
}

template <typename T>
__global__ void ConcatKernel_(const T* input_addr0,
                              const T* input_addr1,
                              const int64_t fixed_in_col,
                              const int64_t out_rows,
                              const int64_t out_cols,
                              T* output_data) {
  const T* inputs_data[2];
  inputs_data[0] = input_addr0;
  inputs_data[1] = input_addr1;
  ConcatKernelDetail<T>(
      inputs_data, fixed_in_col, out_rows, out_cols, output_data);
}

template <typename T>
__global__ void ConcatKernel_(const T* input_addr0,
                              const T* input_addr1,
                              const T* input_addr2,
                              const int64_t fixed_in_col,
                              const int64_t out_rows,
                              const int64_t out_cols,
                              T* output_data) {
  const T* inputs_data[3];
  inputs_data[0] = input_addr0;
  inputs_data[1] = input_addr1;
  inputs_data[2] = input_addr2;
  ConcatKernelDetail<T>(
      inputs_data, fixed_in_col, out_rows, out_cols, output_data);
}

template <typename T>
__global__ void ConcatKernel_(const T* input_addr0,
                              const T* input_addr1,
                              const T* input_addr2,
                              const T* input_addr3,
                              const int64_t fixed_in_col,
                              const int64_t out_rows,
                              const int64_t out_cols,
                              T* output_data) {
  const T* inputs_data[4];
  inputs_data[0] = input_addr0;
  inputs_data[1] = input_addr1;
  inputs_data[2] = input_addr2;
  inputs_data[3] = input_addr3;
  ConcatKernelDetail<T>(
      inputs_data, fixed_in_col, out_rows, out_cols, output_data);
}

template <typename T>
__global__ void ConcatKernel_(const T** inputs_data,
                              const int in_num,
                              const int64_t fixed_in_col,
                              const int64_t out_rows,
                              const int64_t out_cols,
                              T* output_data) {
  ConcatKernelDetail<T>(
      inputs_data, fixed_in_col, out_rows, out_cols, output_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t* out_cols,
                             int out_cols_size,
                             T** outputs_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  int curr_segment = 0;
  int curr_offset = out_cols[0];
  for (; tid_x < in_col; tid_x += blockDim.x * gridDim.x) {
    int curr_col_offset = out_cols[curr_segment + 1];
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = out_cols[curr_segment + 1];
    }

    int local_col = tid_x - curr_offset;
    int segment_width = curr_col_offset - curr_offset;
    T* output_ptr = outputs_data[curr_segment];
    if (output_ptr != nullptr) {
      int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * segment_width + local_col] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__device__ void SplitKernelDetail(const T* input_data,
                                  const int in_row,
                                  const int in_col,
                                  const int fixed_out_col,
                                  T** outputs_data) {
  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  for (; tid_x < in_col; tid_x += blockDim.x * gridDim.x) {
    int split = tid_x / fixed_out_col;
    int in_offset = tid_x - split * fixed_out_col;
    T* output_ptr = outputs_data[split];
    if (output_ptr != nullptr) {
      int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * fixed_out_col + in_offset] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T** outputs_data) {
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1) {
  T* outputs_data[2];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1,
                             T* outputs_addr2) {
  T* outputs_data[3];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  outputs_data[2] = outputs_addr2;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1,
                             T* outputs_addr2,
                             T* outputs_addr3) {
  T* outputs_data[4];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  outputs_data[2] = outputs_addr2;
  outputs_data[3] = outputs_addr3;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

static inline void GetBlockDims(const phi::GPUContext& context,
                                int64_t num_rows,
                                int64_t num_cols,
                                dim3* block_dims,
                                dim3* grid_dims) {
  // Set the thread block and grid according to CurrentDeviceId
  const int kThreadsPerBlock = 1024;
  int block_cols = kThreadsPerBlock;
  if (num_cols < kThreadsPerBlock) {  // block_cols is aligned by 32.
    block_cols = ((num_cols + 31) >> 5) << 5;
  }
  int block_rows = kThreadsPerBlock / block_cols;
  *block_dims = dim3(block_cols, block_rows, 1);

  int max_threads = context.GetMaxPhysicalThreadCount();
  int64_t max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

  int grid_cols =
      std::min((num_cols + block_cols - 1) / block_cols, max_blocks);
  int grid_rows = std::min(max_blocks / grid_cols,
                           std::max(num_rows / block_rows, (int64_t)1));
  *grid_dims = dim3(grid_cols, grid_rows, 1);
}

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */

template <typename T>
struct ConcatFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const std::vector<phi::DenseTensor>& input,
                  int axis,
                  phi::DenseTensor* output) {
    // TODO(zcd): Add input data validity checking
    int in_num = input.size();
    int64_t in_row = 1;
    auto dim_0 = input[0].dims();
    for (int i = 0; i < axis; ++i) {
      in_row *= dim_0[i];
    }
    int64_t in_col = input[0].numel() / in_row;
    int64_t out_row = in_row, out_col = 0;

    int inputs_col_num = in_num + 1;
    std::vector<const T*> inputs_data_vec(in_num);
    std::vector<int64_t> inputs_col_vec(inputs_col_num);
    const T** inputs_data = inputs_data_vec.data();
    int64_t* inputs_col = inputs_col_vec.data();

// There are some differences between hip runtime and NV runtime.
// In NV, when the pageable memory data less than 64K is transferred from
// hosttodevice, it will be automatically asynchronous.
// However, only pinned memory in hip can copy asynchronously
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#concurrent-execution-host-device
// 3.2.6.1. Concurrent Execution between Host and Device
// Memory copies from host to device of a memory block of 64 KB or less
#ifdef PADDLE_WITH_HIP
    paddle::memory::AllocationPtr data_alloc, col_alloc;
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    data_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       in_num * sizeof(T*));
    inputs_data = reinterpret_cast<const T**>(data_alloc->ptr());
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    col_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                      inputs_col_num * sizeof(int));
    inputs_col = reinterpret_cast<int64_t*>(col_alloc->ptr());
#endif

    inputs_col[0] = 0;
    bool has_same_shape = true;
    for (int i = 0; i < in_num; ++i) {
      int64_t t_cols = input[i].numel() / in_row;
      if (has_same_shape) {
        if (t_cols != in_col) has_same_shape = false;
      }
      out_col += t_cols;
      inputs_col[i + 1] = out_col;
      inputs_data[i] = input[i].data<T>();
    }

    dim3 block_dims;
    dim3 grid_dims;
    GetBlockDims(context, out_row, out_col, &block_dims, &grid_dims);

    paddle::memory::allocation::AllocationPtr tmp_dev_ins_data;
    const T** dev_ins_data = nullptr;
    if (!has_same_shape || in_num < 2 || in_num > 4) {
      tmp_dev_ins_data = paddle::memory::Alloc(context, in_num * sizeof(T*));
      auto* restored = paddle::platform::RestoreHostMemIfCapturingCUDAGraph(
          inputs_data, in_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_ins_data->ptr(),
                           paddle::platform::CPUPlace(),
                           restored,
                           in_num * sizeof(T*),
                           context.stream());
      dev_ins_data = reinterpret_cast<const T**>(tmp_dev_ins_data->ptr());
    }

    if (has_same_shape) {
      if (in_num == 2) {
        ConcatKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            inputs_data[0],
            inputs_data[1],
            in_col,
            out_row,
            out_col,
            output->data<T>());
      } else if (in_num == 3) {
        ConcatKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            inputs_data[0],
            inputs_data[1],
            inputs_data[2],
            in_col,
            out_row,
            out_col,
            output->data<T>());
      } else if (in_num == 4) {
        ConcatKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            inputs_data[0],
            inputs_data[1],
            inputs_data[2],
            inputs_data[3],
            in_col,
            out_row,
            out_col,
            output->data<T>());
      } else {
        ConcatKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            dev_ins_data, in_num, in_col, out_row, out_col, output->data<T>());
      }
    } else {
      auto tmp_dev_ins_col_data =
          paddle::memory::Alloc(context, inputs_col_num * sizeof(int64_t));

      auto* restored = paddle::platform::RestoreHostMemIfCapturingCUDAGraph(
          inputs_col, inputs_col_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_ins_col_data->ptr(),
                           paddle::platform::CPUPlace(),
                           restored,
                           inputs_col_num * sizeof(int64_t),
                           context.stream());
      int64_t* dev_ins_col_data =
          static_cast<int64_t*>(tmp_dev_ins_col_data->ptr());

      ConcatKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
          dev_ins_data,
          dev_ins_col_data,
          static_cast<int>(inputs_col_num),
          out_row,
          out_col,
          output->data<T>());
    }

#ifdef PADDLE_WITH_HIP
    // Prevent the pinned memory value from being covered and release the memory
    // after the launch kernel of the stream is executed (reapply pinned memory
    // next time)
    auto* data_alloc_released = data_alloc.release();
    auto* col_alloc_released = col_alloc.release();
    context.AddStreamCallback([data_alloc_released, col_alloc_released] {
      paddle::memory::allocation::Allocator::AllocationDeleter(
          data_alloc_released);
      paddle::memory::allocation::Allocator::AllocationDeleter(
          col_alloc_released);
    });
#endif
  }
};

template <typename T>
class SplitFunctor<phi::GPUContext, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const phi::DenseTensor& input,
                  const std::vector<const phi::DenseTensor*>& ref_inputs,
                  int axis,
                  std::vector<phi::DenseTensor*>* outputs) {
    // NOTE(zhiqiu): split a tensor of shape [0,3,4] at axis=1, result in 3
    // tensors of shape [0,1,4]
    if (input.numel() == 0) {
      return;
    }

    // TODO(zcd): Add input data validity checking
    int o_num = outputs->size();
    int64_t out_row = 1;
    auto dim_0 = ref_inputs[0]->dims();
    for (int i = 0; i < axis; ++i) {
      out_row *= dim_0[i];
    }

    int64_t out0_col = ref_inputs[0]->numel() / out_row;
    int64_t in_col = 0, in_row = out_row;
    bool has_same_shape = true;

    int outputs_cols_num = o_num + 1;
    std::vector<T*> outputs_data_vec(o_num);
    std::vector<int64_t> outputs_cols_vec(outputs_cols_num);
    T** outputs_data = outputs_data_vec.data();
    int64_t* outputs_cols = outputs_cols_vec.data();

// There are some differences between hip runtime and NV runtime.
// In NV, when the pageable memory data less than 64K is transferred from
// hosttodevice, it will be automatically asynchronous.
// However, only pinned memory in hip can copy asynchronously
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#concurrent-execution-host-device
// 3.2.6.1. Concurrent Execution between Host and Device
// Memory copies from host to device of a memory block of 64 KB or less
#ifdef PADDLE_WITH_HIP
    paddle::memory::AllocationPtr data_alloc, cols_alloc;
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    data_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       o_num * sizeof(T*));
    outputs_data = reinterpret_cast<T**>(data_alloc->ptr());
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    cols_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       (outputs_cols_num) * sizeof(int64_t));
    outputs_cols = reinterpret_cast<int64_t*>(cols_alloc->ptr());
#endif

    outputs_cols[0] = 0;
    for (int i = 0; i < o_num; ++i) {
      int64_t t_col = ref_inputs.at(i)->numel() / out_row;
      if (has_same_shape) {
        if (t_col != out0_col) has_same_shape = false;
      }
      in_col += t_col;
      outputs_cols[i + 1] = in_col;
      if (outputs->at(i) != nullptr) {
        outputs_data[i] = outputs->at(i)->data<T>();
      } else {
        outputs_data[i] = nullptr;
      }
    }

    dim3 block_dims;
    dim3 grid_dims;
    GetBlockDims(context, out_row, in_col, &block_dims, &grid_dims);

    paddle::memory::allocation::AllocationPtr tmp_dev_outs_data;
    T** dev_out_gpu_data = nullptr;
    if (!has_same_shape || o_num < 2 || o_num > 4) {
      // TODO(chentianyu03): try to find a method to remove the Alloc function
      tmp_dev_outs_data = paddle::memory::Alloc(context, o_num * sizeof(T*));
      auto* restored = paddle::platform::RestoreHostMemIfCapturingCUDAGraph(
          outputs_data, o_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_outs_data->ptr(),
                           paddle::platform::CPUPlace(),
                           restored,
                           o_num * sizeof(T*),
                           context.stream());
      dev_out_gpu_data = reinterpret_cast<T**>(tmp_dev_outs_data->ptr());
    }

    if (has_same_shape) {
      if (o_num == 2) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1]);
      } else if (o_num == 3) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1],
            outputs_data[2]);
      } else if (o_num == 4) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1],
            outputs_data[2],
            outputs_data[3]);
      } else {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(), in_row, in_col, out0_col, dev_out_gpu_data);
      }
    } else {
      auto tmp_dev_ins_col_data =
          // TODO(chentianyu03): try to find a method to remove the Alloc
          // function
          paddle::memory::Alloc(context, outputs_cols_num * sizeof(int64_t));
      auto* restored = paddle::platform::RestoreHostMemIfCapturingCUDAGraph(
          outputs_cols, outputs_cols_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_ins_col_data->ptr(),
                           paddle::platform::CPUPlace(),
                           restored,
                           outputs_cols_num * sizeof(int64_t),
                           context.stream());
      int64_t* dev_outs_col_data =
          reinterpret_cast<int64_t*>(tmp_dev_ins_col_data->ptr());

      SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
          input.data<T>(),
          in_row,
          in_col,
          dev_outs_col_data,
          static_cast<int>(outputs_cols_num),
          dev_out_gpu_data);
    }
#ifdef PADDLE_WITH_HIP
    // Prevent the pinned memory value from being covered and release the memory
    // after the launch kernel of the stream is executed (reapply pinned memory
    // next time)
    auto* data_alloc_released = data_alloc.release();
    auto* cols_alloc_released = cols_alloc.release();
    context.AddStreamCallback([data_alloc_released, cols_alloc_released] {
      paddle::memory::allocation::Allocator::AllocationDeleter(
          data_alloc_released);
      paddle::memory::allocation::Allocator::AllocationDeleter(
          cols_alloc_released);
    });
#endif
  }
};

#define DEFINE_FUNCTOR(type)                           \
  template class ConcatFunctor<phi::GPUContext, type>; \
  template class SplitFunctor<phi::GPUContext, type>

FOR_ALL_TYPES(DEFINE_FUNCTOR);

}  // namespace funcs
}  // namespace phi
