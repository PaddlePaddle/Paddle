#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/funcs/concat_and_split_functor.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/phi/backends/gpu/cuda/cuda_graph_with_memory_pool.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"

namespace phi {
namespace funcs {

static inline void GetBlockDims(const phi::GPUContext& context,
                                int64_t num_rows,
                                int64_t num_cols,
                                dim3* block_dims,
                                dim3* grid_dims) {
  // Set the thread block and grid according to CurrentDeviceId
  const int kThreadsPerBlock = 1024;
  int block_cols = kThreadsPerBlock;
  if (num_cols < kThreadsPerBlock) {  // block_cols is aligned by 32.
    block_cols = ((num_cols + 31) >> 5) << 5;
  }
  int block_rows = kThreadsPerBlock / block_cols;
  *block_dims = dim3(block_cols, block_rows, 1);

  constexpr int waves = 1;
  int max_threads = context.GetMaxPhysicalThreadCount() * waves;
  int64_t max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

  int grid_cols =
      std::min((num_cols + block_cols - 1) / block_cols, max_blocks);
  int grid_rows = std::min(max_blocks / grid_cols,
                           std::max(num_rows / block_rows, (int64_t)1));
  *grid_dims = dim3(grid_cols, grid_rows, 1);
}

template <typename T, int Size>
struct PointerWrapper {
 public:
  const void* ins_addr[Size];
  __device__ inline const void* operator[](int i) const { return ins_addr[i]; }

  PointerWrapper() {}
  PointerWrapper(const phi::GPUContext& ctx,
                 const std::vector<phi::DenseTensor>& ins,
                 const T** pre_alloced_host_ptr) {
    for (auto i = 0; i < ins.size(); ++i) {
      ins_addr[i] = ins[i].data();
    }
  }
};

template <typename T>
struct PointerToPointer {
 public:
  void** ins_addr{nullptr};
  __device__ inline const void* operator[](int i) const { return ins_addr[i]; }

  PointerToPointer() {}
  PointerToPointer(const phi::GPUContext& ctx,
                   const std::vector<phi::DenseTensor>& ins,
                   const T** pre_alloced_host_ptr,
                   paddle::memory::AllocationPtr* dev_ins_ptr) {
    auto in_num = ins.size();
    for (auto i = 0; i < in_num; ++i) {
      pre_alloced_host_ptr[i] = ins[i].data<T>();
    }
    *dev_ins_ptr = paddle::memory::Alloc(
        ctx.GetPlace(),
        in_num * sizeof(T*),
        phi::Stream(reinterpret_cast<phi::StreamId>(ctx.stream())));
    auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
        pre_alloced_host_ptr, in_num);
    paddle::memory::Copy(ctx.GetPlace(),
                         (*dev_ins_ptr)->ptr(),
                         phi::CPUPlace(),
                         restored,
                         in_num * sizeof(T*),
                         ctx.stream());
    ins_addr = reinterpret_cast<void**>((*dev_ins_ptr)->ptr());
  }
};

template <typename T, typename IndexT, int Size>
struct PointerAndColWrapper {
 public:
  IndexT col_length[Size];
  PointerAndColWrapper(const phi::GPUContext& ctx,
                       const std::vector<phi::DenseTensor>& ins,
                       const IndexT& inputs_col_num,
                       const T** pre_alloced_host_ptr,
                       IndexT* inputs_col) {
    for (auto i = 0; i < inputs_col_num; ++i) {
      col_length[i] = inputs_col[i];
    }
    ins_ptr_wrapper = PointerWrapper<T, Size>(ctx, ins, pre_alloced_host_ptr);
  }

  __device__ inline const void* operator[](int i) const {
    return ins_ptr_wrapper[i];
  }

 private:
  PointerWrapper<T, Size> ins_ptr_wrapper;
};

template <typename T, typename IndexT>
struct PointerToPointerAndCol {
 public:
  IndexT* col_length{nullptr};
  PointerToPointerAndCol(const phi::GPUContext& ctx,
                         const std::vector<phi::DenseTensor>& ins,
                         const IndexT inputs_col_num,
                         const T** pre_alloced_host_ptr,
                         IndexT* inputs_col,
                         paddle::memory::AllocationPtr* dev_ins_ptr,
                         paddle::memory::AllocationPtr* dev_col_ptr) {
    *dev_col_ptr = paddle::memory::Alloc(
        ctx.GetPlace(),
        inputs_col_num * sizeof(IndexT),
        phi::Stream(reinterpret_cast<phi::StreamId>(ctx.stream())));
    auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
        inputs_col, inputs_col_num);
    paddle::memory::Copy(ctx.GetPlace(),
                         (*dev_col_ptr)->ptr(),
                         phi::CPUPlace(),
                         restored,
                         inputs_col_num * sizeof(IndexT),
                         ctx.stream());
    col_length = static_cast<IndexT*>((*dev_col_ptr)->ptr());
    ins_ptr_wrapper =
        PointerToPointer<T>(ctx, ins, pre_alloced_host_ptr, dev_ins_ptr);
  }

  __device__ inline const void* operator[](int i) const {
    return ins_ptr_wrapper[i];
  }

 private:
  PointerToPointer<T> ins_ptr_wrapper;
};

template <typename IndexT, int MovSize, typename PointerAndColWrapperT>
__global__ void ConcatTensorWithDifferentShape(
    const PointerAndColWrapperT ins_datas,
    int col_size,
    const IndexT output_rows,
    const IndexT output_cols,
    void* output) {
  using VecT = typename std::aligned_storage<MovSize, MovSize>::type;
  VecT* dst = reinterpret_cast<VecT*>(output);

  IndexT curr_segment = 0;
  IndexT curr_offset = ins_datas.col_length[0];

  CUDA_KERNEL_LOOP_TYPE(tid_x, output_cols, IndexT) {
    IndexT curr_col_offset = ins_datas.col_length[curr_segment + 1];

    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = ins_datas.col_length[curr_segment + 1];
    }

    IndexT local_col = tid_x - curr_offset;
    IndexT segment_width = curr_col_offset - curr_offset;

    const VecT* input_ptr =
        reinterpret_cast<const VecT*>(ins_datas[curr_segment]);

    IndexT tid_y = blockIdx.y * blockDim.y + threadIdx.y;

    for (; tid_y < output_rows; tid_y += blockDim.y * gridDim.y) {
      dst[tid_y * output_cols + tid_x] =
          input_ptr[tid_y * segment_width + local_col];
    }
  }
}

template <typename IndexT, int MovSize, typename PointerWrapperT>
__global__ void ConcatTensorWithSameShape(const PointerWrapperT ins_data,
                                          const IndexT fixed_in_col,
                                          const IndexT out_rows,
                                          const IndexT out_cols,
                                          void* output_data) {
  using VecT = typename std::aligned_storage<MovSize, MovSize>::type;
  VecT* dst = reinterpret_cast<VecT*>(output_data);
  CUDA_KERNEL_LOOP_TYPE(tid_x, out_cols, IndexT) {
    IndexT split = tid_x / fixed_in_col;
    IndexT in_offset = tid_x - split * fixed_in_col;
    const VecT* input_ptr = reinterpret_cast<const VecT*>(ins_data[split]);
    IndexT tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < out_rows; tid_y += blockDim.y * gridDim.y) {
      dst[tid_y * out_cols + tid_x] =
          input_ptr[tid_y * fixed_in_col + in_offset];
    }
  }
}

#define IMPL_CONCATE_CUDA_KERNEL_HELPER(func_impl, ...) \
  func_impl(4, ##__VA_ARGS__);                          \
  func_impl(8, ##__VA_ARGS__);                          \
  func_impl(16, ##__VA_ARGS__);                         \
  func_impl(32, ##__VA_ARGS__);                         \
  func_impl(64, ##__VA_ARGS__);                         \
  func_impl(128, ##__VA_ARGS__);

template <typename T, typename IndexT, int MovSize>
void DispatchConcatWithDifferentShapeKernelLimitNum(
    const phi::GPUContext& ctx,
    const std::vector<phi::DenseTensor>& ins,
    const IndexT inputs_col_num,
    const T** inputs_data,
    IndexT* inputs_col,
    const IndexT out_row,
    const IndexT out_col,
    phi::DenseTensor* output,
    const IndexT in_num,
    const IndexT limit_num) {
  dim3 block_dims;
  dim3 grid_dims;
  GetBlockDims(ctx, out_row, out_col, &block_dims, &grid_dims);

#define IMPL_COMPLEX_CONCAT_CUDA_KERNEL_CASE(size_, ...)    \
  case size_: {                                             \
    PointerAndColWrapper<T, IndexT, size_> ptr_col_array(   \
        ctx, ins, inputs_col_num, inputs_data, inputs_col); \
    __VA_ARGS__;                                            \
  } break;
  switch (phi::backends::gpu::RoundToNextHighPowOfTwo(limit_num, 4)) {
    IMPL_CONCATE_CUDA_KERNEL_HELPER(
        IMPL_COMPLEX_CONCAT_CUDA_KERNEL_CASE,
        ConcatTensorWithDifferentShape<IndexT, MovSize, decltype(ptr_col_array)>
        <<<grid_dims, block_dims, 0, ctx.stream()>>>(
            ptr_col_array, inputs_col_num, out_row, out_col, output->data()));
    default: {
      paddle::memory::AllocationPtr dev_ins_ptr{nullptr};
      paddle::memory::AllocationPtr dev_col_ptr{nullptr};
      PointerToPointerAndCol<T, IndexT> ptr_col_array(ctx,
                                                      ins,
                                                      inputs_col_num,
                                                      inputs_data,
                                                      inputs_col,
                                                      &dev_ins_ptr,
                                                      &dev_col_ptr);
      ConcatTensorWithDifferentShape<IndexT, MovSize, decltype(ptr_col_array)>
          <<<grid_dims, block_dims, 0, ctx.stream()>>>(
              ptr_col_array, inputs_col_num, out_row, out_col, output->data());
    }
  }
#undef IMPL_COMPLEX_CONCAT_CUDA_KERNEL_CASE
}

template <typename T, typename IndexT>
void DispatchConcatWithDifferentShapeMovsize(
    const phi::GPUContext& ctx,
    const std::vector<phi::DenseTensor>& ins,
    const IndexT inputs_col_num,
    const T** inputs_data,
    IndexT* inputs_col,
    const IndexT out_row,
    const IndexT out_col,
    phi::DenseTensor* output,
    const IndexT mov_size,
    const IndexT in_num,
    const IndexT limit_num) {
  if (mov_size == 16) {
    DispatchConcatWithDifferentShapeKernelLimitNum<T, IndexT, 16>(
        ctx,
        ins,
        inputs_col_num,
        inputs_data,
        inputs_col,
        out_row,
        out_col,
        output,
        in_num,
        limit_num);
  } else if (mov_size == 8) {
    DispatchConcatWithDifferentShapeKernelLimitNum<T, IndexT, 8>(ctx,
                                                                 ins,
                                                                 inputs_col_num,
                                                                 inputs_data,
                                                                 inputs_col,
                                                                 out_row,
                                                                 out_col,
                                                                 output,
                                                                 in_num,
                                                                 limit_num);
  } else if (mov_size == 4) {
    DispatchConcatWithDifferentShapeKernelLimitNum<T, IndexT, 4>(ctx,
                                                                 ins,
                                                                 inputs_col_num,
                                                                 inputs_data,
                                                                 inputs_col,
                                                                 out_row,
                                                                 out_col,
                                                                 output,
                                                                 in_num,
                                                                 limit_num);
  } else if (mov_size == 2) {
    DispatchConcatWithDifferentShapeKernelLimitNum<T, IndexT, 2>(ctx,
                                                                 ins,
                                                                 inputs_col_num,
                                                                 inputs_data,
                                                                 inputs_col,
                                                                 out_row,
                                                                 out_col,
                                                                 output,
                                                                 in_num,
                                                                 limit_num);
  } else {
    DispatchConcatWithDifferentShapeKernelLimitNum<T, IndexT, 1>(ctx,
                                                                 ins,
                                                                 inputs_col_num,
                                                                 inputs_data,
                                                                 inputs_col,
                                                                 out_row,
                                                                 out_col,
                                                                 output,
                                                                 in_num,
                                                                 limit_num);
  }
}

template <typename T, typename IndexT, int MovSize>
void DispatchConcatWithSameShapeKernelLimitNum(
    const phi::GPUContext& ctx,
    const std::vector<phi::DenseTensor>& ins,
    const T** inputs_data,
    IndexT in_col,
    const IndexT out_row,
    const IndexT out_col,
    phi::DenseTensor* output,
    const IndexT in_num,
    const IndexT limit_num) {
  dim3 block_dims;
  dim3 grid_dims;
  GetBlockDims(ctx, out_row, out_col, &block_dims, &grid_dims);

#define IMPL_CONCAT_CUDA_KERNEL_CASE(size_, ...)               \
  case size_: {                                                \
    PointerWrapper<T, size_> ptr_array(ctx, ins, inputs_data); \
    __VA_ARGS__;                                               \
  } break;

  switch (phi::backends::gpu::RoundToNextHighPowOfTwo(limit_num, 4)) {
    IMPL_CONCATE_CUDA_KERNEL_HELPER(
        IMPL_CONCAT_CUDA_KERNEL_CASE,
        ConcatTensorWithSameShape<IndexT, MovSize, decltype(ptr_array)>
        <<<grid_dims, block_dims, 0, ctx.stream()>>>(
            ptr_array, in_col, out_row, out_col, output->data()));
    default: {
      paddle::memory::AllocationPtr dev_ins_ptr{nullptr};
      PointerToPointer<T> ptr_array(ctx, ins, inputs_data, &dev_ins_ptr);
      ConcatTensorWithSameShape<IndexT, MovSize, decltype(ptr_array)>
          <<<grid_dims, block_dims, 0, ctx.stream()>>>(
              ptr_array, in_col, out_row, out_col, output->data());
    }
  }
#undef IMPL_CONCAT_CUDA_KERNEL_CASE
}

#undef IMPL_CONCATE_CUDA_KERNEL_HELPER

template <typename T, typename IndexT>
void DispatchConcatWithSameShapeMovsize(
    const phi::GPUContext& ctx,
    const std::vector<phi::DenseTensor>& ins,
    const T** inputs_data,
    IndexT in_col,
    const IndexT out_row,
    const IndexT out_col,
    phi::DenseTensor* output,
    const IndexT mov_size,
    const IndexT in_num,
    const IndexT limit_num) {
  if (mov_size == 16) {
    DispatchConcatWithSameShapeKernelLimitNum<T, IndexT, 16>(ctx,
                                                             ins,
                                                             inputs_data,
                                                             in_col,
                                                             out_row,
                                                             out_col,
                                                             output,
                                                             in_num,
                                                             limit_num);
  } else if (mov_size == 8) {
    DispatchConcatWithSameShapeKernelLimitNum<T, IndexT, 8>(ctx,
                                                            ins,
                                                            inputs_data,
                                                            in_col,
                                                            out_row,
                                                            out_col,
                                                            output,
                                                            in_num,
                                                            limit_num);
  } else if (mov_size == 4) {
    DispatchConcatWithSameShapeKernelLimitNum<T, IndexT, 4>(ctx,
                                                            ins,
                                                            inputs_data,
                                                            in_col,
                                                            out_row,
                                                            out_col,
                                                            output,
                                                            in_num,
                                                            limit_num);
  } else if (mov_size == 2) {
    DispatchConcatWithSameShapeKernelLimitNum<T, IndexT, 2>(ctx,
                                                            ins,
                                                            inputs_data,
                                                            in_col,
                                                            out_row,
                                                            out_col,
                                                            output,
                                                            in_num,
                                                            limit_num);
  } else {
    DispatchConcatWithSameShapeKernelLimitNum<T, IndexT, 1>(ctx,
                                                            ins,
                                                            inputs_data,
                                                            in_col,
                                                            out_row,
                                                            out_col,
                                                            output,
                                                            in_num,
                                                            limit_num);
  }
}

template <typename T, typename IndexT>
void DispatchConcatKernel(const phi::GPUContext& ctx,
                          const std::vector<phi::DenseTensor>& ins,
                          const IndexT inputs_col_num,
                          const T** inputs_data,
                          IndexT* inputs_col,
                          const IndexT out_row,
                          const IndexT out_col,
                          phi::DenseTensor* output,
                          const IndexT in_num,
                          const IndexT limit_num,
                          bool has_same_shape) {
  constexpr IndexT MaxVecSize = 16 / sizeof(T);
  bool find_vecsize_flag = false;
  IndexT dispatch_vec_size = 1;
  for (IndexT vec_size = MaxVecSize; vec_size > 0; vec_size /= 2) {
    for (IndexT idx = 0; idx < in_num + 1; idx++) {
      // Since input_cols[0] is 0, we need to jump.
      const IndexT input_col = inputs_col[idx + 1] - inputs_col[idx];
      if (input_col % vec_size == 0) {
        if (idx == in_num - 1) {
          find_vecsize_flag = true;
        }
      } else {
        break;
      }
    }
    if (find_vecsize_flag) {
      dispatch_vec_size = vec_size;
      break;
    }
  }

  const int64_t vectorized_out_col = out_col / dispatch_vec_size;
  for (IndexT idx = 0; idx < in_num + 1; idx++) {
    inputs_col[idx] /= dispatch_vec_size;
  }
  const IndexT mov_size = sizeof(T) * dispatch_vec_size;
  if (has_same_shape) {
    // In same shape situation, each input's col are equal, so here we select to
    // use inputs_col[1].
    DispatchConcatWithSameShapeMovsize<T, IndexT>(ctx,
                                                  ins,
                                                  inputs_data,
                                                  inputs_col[1],
                                                  out_row,
                                                  vectorized_out_col,
                                                  output,
                                                  mov_size,
                                                  in_num,
                                                  limit_num);
  } else {
    DispatchConcatWithDifferentShapeMovsize<T, IndexT>(ctx,
                                                       ins,
                                                       inputs_col_num,
                                                       inputs_data,
                                                       inputs_col,
                                                       out_row,
                                                       vectorized_out_col,
                                                       output,
                                                       mov_size,
                                                       in_num,
                                                       limit_num);
  }
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t* out_cols,
                             int out_cols_size,
                             T** outputs_data) {
  int64_t curr_segment = 0;
  int64_t curr_offset = out_cols[0];
  CUDA_KERNEL_LOOP_TYPE(tid_x, in_col, int64_t) {
    int64_t curr_col_offset = out_cols[curr_segment + 1];
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = out_cols[curr_segment + 1];
    }

    int64_t local_col = tid_x - curr_offset;
    int64_t segment_width = curr_col_offset - curr_offset;
    T* output_ptr = outputs_data[curr_segment];
    if (output_ptr != nullptr) {
      int64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * segment_width + local_col] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__device__ void SplitKernelDetail(const T* input_data,
                                  const int64_t in_row,
                                  const int64_t in_col,
                                  const int64_t fixed_out_col,
                                  T** outputs_data) {
  CUDA_KERNEL_LOOP_TYPE(tid_x, in_col, int64_t) {
    int64_t split = tid_x / fixed_out_col;
    int64_t in_offset = tid_x - split * fixed_out_col;
    T* output_ptr = outputs_data[split];
    if (output_ptr != nullptr) {
      int64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * fixed_out_col + in_offset] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T** outputs_data) {
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1) {
  T* outputs_data[2];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1,
                             T* outputs_addr2) {
  T* outputs_data[3];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  outputs_data[2] = outputs_addr2;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1,
                             T* outputs_addr2,
                             T* outputs_addr3) {
  T* outputs_data[4];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  outputs_data[2] = outputs_addr2;
  outputs_data[3] = outputs_addr3;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */
template <typename T, typename IndexT>
void ConcatFunctorWithIndexType(const phi::GPUContext& ctx,
                                const std::vector<phi::DenseTensor>& ins,
                                int axis,
                                phi::DenseTensor* output) {
  // TODO(zcd): Add input data validity checking
  IndexT in_num = ins.size();
  IndexT in_row = 1;
  auto dim_0 = ins[0].dims();
  for (int i = 0; i < axis; ++i) {
    in_row *= dim_0[i];
  }
  IndexT in_col = ins[0].numel() / in_row;
  IndexT out_row = in_row, out_col = 0;

  IndexT inputs_col_num = in_num + 1;
  std::vector<const T*> inputs_data_vec(in_num, nullptr);
  std::vector<IndexT> inputs_col_vec(inputs_col_num, 0);
  const T** inputs_data = inputs_data_vec.data();
  IndexT* inputs_col = inputs_col_vec.data();
#ifdef PADDLE_WITH_HIP
  // TODO(chentianyu03): try to find a method to remove the Alloc function
  paddle::memory::AllocationPtr data_alloc = paddle::memory::Alloc(
      paddle::platform::CUDAPinnedPlace(), in_num * sizeof(T*));
  inputs_data = reinterpret_cast<const T**>(data_alloc->ptr());
  paddle::memory::AllocationPtr col_alloc = paddle::memory::Alloc(
      paddle::platform::CUDAPinnedPlace(), inputs_col_num * sizeof(IndexT));
  inputs_col = reinterpret_cast<IndexT*>(col_alloc->ptr());
#endif

  bool has_same_shape = true;
  for (int i = 0; i < in_num; ++i) {
    IndexT t_cols = ins[i].numel() / in_row;
    if (has_same_shape) {
      has_same_shape &= (t_cols == in_col);
    }
    out_col += t_cols;
    inputs_col[i + 1] = out_col;
  }
  IndexT limit_num = has_same_shape ? in_num : inputs_col_num;

  DispatchConcatKernel<T, IndexT>(ctx,
                                  ins,
                                  inputs_col_num,
                                  inputs_data,
                                  inputs_col,
                                  out_row,
                                  out_col,
                                  output,
                                  in_num,
                                  limit_num,
                                  has_same_shape);

#ifdef PADDLE_WITH_HIP
  // Prevent pinned memory from being covered and release the memory after
  // kernel launch of the stream is executed (reapply pinned memory next time)
  auto* data_alloc_released = data_alloc.release();
  auto* col_alloc_released = col_alloc.release();
  ctx.AddStreamCallback([data_alloc_released, col_alloc_released] {
    VLOG(4) << "Delete cuda pinned at " << data_alloc_released;
    VLOG(4) << "Delete cuda pinned at " << col_alloc_released;
    paddle::memory::allocation::Allocator::AllocationDeleter(
        data_alloc_released);
    paddle::memory::allocation::Allocator::AllocationDeleter(
        col_alloc_released);
  });
#endif
}

template <typename T>
struct ConcatFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const std::vector<phi::DenseTensor>& input,
                  int axis,
                  phi::DenseTensor* output) {
    if (output->numel() < std::numeric_limits<int32_t>::max()) {
      ConcatFunctorWithIndexType<T, int32_t>(context, input, axis, output);
    } else {
      ConcatFunctorWithIndexType<T, int64_t>(context, input, axis, output);
    }
  }
};

template <typename T>
class SplitFunctor<phi::GPUContext, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const phi::DenseTensor& input,
                  const std::vector<const phi::DenseTensor*>& ref_inputs,
                  int axis,
                  std::vector<phi::DenseTensor*>* outputs) {
    // NOTE(zhiqiu): split a tensor of shape [0,3,4] at axis=1, result in 3
    // tensors of shape [0,1,4]
    if (input.numel() == 0) {
      return;
    }

    // TODO(zcd): Add input data validity checking
    int o_num = outputs->size();
    int64_t out_row = 1;
    auto dim_0 = ref_inputs[0]->dims();
    for (int i = 0; i < axis; ++i) {
      out_row *= dim_0[i];
    }

    int64_t out0_col = ref_inputs[0]->numel() / out_row;
    int64_t in_col = 0, in_row = out_row;
    bool has_same_shape = true;

    int outputs_cols_num = o_num + 1;
    std::vector<T*> outputs_data_vec(o_num);
    std::vector<int64_t> outputs_cols_vec(outputs_cols_num);
    T** outputs_data = outputs_data_vec.data();
    int64_t* outputs_cols = outputs_cols_vec.data();

// There are some differences between hip runtime and NV runtime.
// In NV, when the pageable memory data less than 64K is transferred from
// hosttodevice, it will be automatically asynchronous.
// However, only pinned memory in hip can copy asynchronously
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#concurrent-execution-host-device
// 3.2.6.1. Concurrent Execution between Host and Device
// Memory copies from host to device of a memory block of 64 KB or less
#ifdef PADDLE_WITH_HIP
    paddle::memory::AllocationPtr data_alloc, cols_alloc;
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    data_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       o_num * sizeof(T*));
    outputs_data = reinterpret_cast<T**>(data_alloc->ptr());
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    cols_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       (outputs_cols_num) * sizeof(int64_t));
    outputs_cols = reinterpret_cast<int64_t*>(cols_alloc->ptr());
#endif

    outputs_cols[0] = 0;
    for (int i = 0; i < o_num; ++i) {
      int64_t t_col = ref_inputs.at(i)->numel() / out_row;
      if (has_same_shape) {
        if (t_col != out0_col) has_same_shape = false;
      }
      in_col += t_col;
      outputs_cols[i + 1] = in_col;
      if (outputs->at(i) != nullptr) {
        outputs_data[i] = outputs->at(i)->data<T>();
      } else {
        outputs_data[i] = nullptr;
      }
    }

    dim3 block_dims;
    dim3 grid_dims;
    GetBlockDims(context, out_row, in_col, &block_dims, &grid_dims);

    paddle::memory::allocation::AllocationPtr tmp_dev_outs_data;
    T** dev_out_gpu_data = nullptr;
    if (!has_same_shape || o_num < 2 || o_num > 4) {
      // TODO(chentianyu03): try to find a method to remove the Alloc function
      tmp_dev_outs_data = paddle::memory::Alloc(
          context.GetPlace(),
          o_num * sizeof(T*),
          phi::Stream(reinterpret_cast<phi::StreamId>(context.stream())));
      auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
          outputs_data, o_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_outs_data->ptr(),
                           phi::CPUPlace(),
                           restored,
                           o_num * sizeof(T*),
                           context.stream());
      dev_out_gpu_data = reinterpret_cast<T**>(tmp_dev_outs_data->ptr());
    }

    if (has_same_shape) {
      if (o_num == 2) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1]);
      } else if (o_num == 3) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1],
            outputs_data[2]);
      } else if (o_num == 4) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1],
            outputs_data[2],
            outputs_data[3]);
      } else {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(), in_row, in_col, out0_col, dev_out_gpu_data);
      }
    } else {
      auto tmp_dev_ins_col_data =
          // TODO(chentianyu03): try to find a method to remove the Alloc
          // function
          paddle::memory::Alloc(
              context.GetPlace(),
              outputs_cols_num * sizeof(int64_t),
              phi::Stream(reinterpret_cast<phi::StreamId>(context.stream())));
      auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
          outputs_cols, outputs_cols_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_ins_col_data->ptr(),
                           phi::CPUPlace(),
                           restored,
                           outputs_cols_num * sizeof(int64_t),
                           context.stream());
      int64_t* dev_outs_col_data =
          reinterpret_cast<int64_t*>(tmp_dev_ins_col_data->ptr());

      SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
          input.data<T>(),
          in_row,
          in_col,
          dev_outs_col_data,
          static_cast<int>(outputs_cols_num),
          dev_out_gpu_data);
    }

#ifdef PADDLE_WITH_HIP
    // Prevent the pinned memory value from being covered and release the memory
    // after the launch kernel of the stream is executed (reapply pinned memory
    // next time)
    auto* data_alloc_released = data_alloc.release();
    auto* cols_alloc_released = cols_alloc.release();
    context.AddStreamCallback([data_alloc_released, cols_alloc_released] {
      paddle::memory::allocation::Allocator::AllocationDeleter(
          data_alloc_released);
      paddle::memory::allocation::Allocator::AllocationDeleter(
          cols_alloc_released);
    });
#endif
  }
};

#define DEFINE_FUNCTOR(type)                           \
  template class ConcatFunctor<phi::GPUContext, type>; \
  template class SplitFunctor<phi::GPUContext, type>

FOR_ALL_TYPES(DEFINE_FUNCTOR);

}  // namespace funcs
}  // namespace phi
