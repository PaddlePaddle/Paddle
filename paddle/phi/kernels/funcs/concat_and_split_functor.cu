#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/funcs/concat_and_split_functor.h"
#include "paddle/fluid/memory/malloc.h"
#include "paddle/phi/backends/gpu/cuda/cuda_graph_with_memory_pool.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"

namespace phi {
namespace funcs {

template <typename T, int Size>
struct PointerWrapper {
 public:
  const T* ins_addr[Size];
  __device__ inline const T* operator[](int i) const { return ins_addr[i]; }

  PointerWrapper() {}
  PointerWrapper(const phi::GPUContext& ctx,
                 const std::vector<phi::DenseTensor>& ins,
                 const T** pre_alloced_host_ptr) {
    for (auto i = 0; i < ins.size(); ++i) {
      ins_addr[i] = ins[i].data<T>();
    }
  }
};

template <typename T>
struct PointerWrapper<T, 0> {
 public:
  T** ins_addr{nullptr};
  __device__ inline const T* operator[](int i) const { return ins_addr[i]; }

  PointerWrapper() {}
  PointerWrapper(const phi::GPUContext& ctx,
                 const std::vector<phi::DenseTensor>& ins,
                 const T** pre_alloced_host_ptr) {
    auto in_num = ins.size();
    for (auto i = 0; i < in_num; ++i) {
      pre_alloced_host_ptr[i] = ins[i].data<T>();
    }
    auto tmp_ins_ptr = paddle::memory::Alloc(
            ctx.GetPlace(),
            in_num * sizeof(T*),
            phi::Stream(reinterpret_cast<phi::StreamId>(ctx.stream())));
    *tmp_dev_ins_ptr = std::move(tmp_ins_ptr);
    auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
        pre_alloced_host_ptr, in_num);
    paddle::memory::Copy(ctx.GetPlace(),
                         (*tmp_dev_ins_ptr)->ptr(),
                         phi::CPUPlace(),
                         restored,
                         in_num * sizeof(T*),
                         ctx.stream());
    ins_addr = reinterpret_cast<T**>((*tmp_dev_ins_ptr)->ptr());
  }
 
 private :
  paddle::memory::AllocationPtr* tmp_dev_ins_ptr{nullptr};
};

template <typename T, typename IndexT, int Size>
struct PointerAndColWrapper {
 public:
  IndexT col_length[Size];
  PointerAndColWrapper(const phi::GPUContext& ctx,
                       const std::vector<phi::DenseTensor>& ins,
                       const IndexT& inputs_col_num,
                       const T** pre_alloced_host_ptr,
                       IndexT* inputs_col) {
    for (auto i = 0; i < inputs_col_num; ++i) {
      col_length[i] = inputs_col[i];
    }
    ins_ptr_wrapper = PointerWrapper<T, Size>(ctx, ins, pre_alloced_host_ptr);
  }

  __device__ inline const T* operator[](int i) const { return ins_ptr_wrapper[i]; }

 private:
  PointerWrapper<T, Size> ins_ptr_wrapper;
};

template <typename T, typename IndexT>
struct PointerAndColWrapper<T, IndexT, 0> {
 public:
  IndexT* col_length;
  PointerAndColWrapper(const phi::GPUContext& ctx,
                       const std::vector<phi::DenseTensor>& ins,
                       const IndexT& inputs_col_num,
                       const T** pre_alloced_host_ptr,
                       IndexT* inputs_col) {
    auto tmp_col_ptr = paddle::memory::Alloc(
            ctx.GetPlace(),
            inputs_col_num * sizeof(IndexT),
            phi::Stream(reinterpret_cast<phi::StreamId>(ctx.stream())));
    *tmp_dev_col_ptr = std::move(tmp_col_ptr);
    auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
        inputs_col, inputs_col_num);
    paddle::memory::Copy(ctx.GetPlace(),
                         (*tmp_dev_col_ptr)->ptr(),
                         phi::CPUPlace(),
                         restored,
                         inputs_col_num * sizeof(IndexT),
                         ctx.stream());
    col_length = static_cast<IndexT*>((*tmp_dev_col_ptr)->ptr());
    ins_ptr_wrapper = PointerWrapper<T, 0>(ctx, ins, pre_alloced_host_ptr);
  }

  __device__ inline const T* operator[](int i) const { return ins_ptr_wrapper[i]; }

 private:
  paddle::memory::AllocationPtr* tmp_dev_col_ptr{nullptr};
  PointerWrapper<T, 0> ins_ptr_wrapper;
};

template <typename T, typename IndexT, typename PointerAndColWrapperT>
__global__ void ConcatTensorWithDifferentShape(PointerAndColWrapperT ins_datas,
                                               int col_size,
                                               const IndexT output_rows,
                                               const IndexT output_cols,
                                               T* output) {
  IndexT curr_segment = 0;
  IndexT curr_offset = ins_datas.col_length[0];
  CUDA_KERNEL_LOOP_TYPE(tid_x, output_cols, IndexT) {
    IndexT curr_col_offset = ins_datas.col_length[curr_segment + 1];
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = ins_datas.col_length[curr_segment + 1];
    }

    IndexT local_col = tid_x - curr_offset;
    IndexT segment_width = curr_col_offset - curr_offset;

    const T* input_ptr = ins_datas[curr_segment];
    IndexT tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < output_rows; tid_y += blockDim.y * gridDim.y)
      output[tid_y * output_cols + tid_x] =
          input_ptr[tid_y * segment_width + local_col];
  }
}

template <typename T, typename IndexT, typename PointerWrapperT>
__global__ void ConcatTensorWithSameShape(PointerWrapperT ins_data,
                                          const IndexT fixed_in_col,
                                          const IndexT out_rows,
                                          const IndexT out_cols,
                                          T* output_data) {
  CUDA_KERNEL_LOOP_TYPE(tid_x, out_cols, IndexT) {
    IndexT split = tid_x / fixed_in_col;
    IndexT in_offset = tid_x - split * fixed_in_col;
    const T* input_ptr = ins_data[split];
    IndexT tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    for (; tid_y < out_rows; tid_y += blockDim.y * gridDim.y) {
      output_data[tid_y * out_cols + tid_x] =
          input_ptr[tid_y * fixed_in_col + in_offset];
    }
  }
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t* out_cols,
                             int out_cols_size,
                             T** outputs_data) {
  int64_t curr_segment = 0;
  int64_t curr_offset = out_cols[0];
  CUDA_KERNEL_LOOP_TYPE(tid_x, in_col, int64_t) {
    int64_t curr_col_offset = out_cols[curr_segment + 1];
    while (curr_col_offset <= tid_x) {
      curr_offset = curr_col_offset;
      ++curr_segment;
      curr_col_offset = out_cols[curr_segment + 1];
    }

    int64_t local_col = tid_x - curr_offset;
    int64_t segment_width = curr_col_offset - curr_offset;
    T* output_ptr = outputs_data[curr_segment];
    if (output_ptr != nullptr) {
      int64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * segment_width + local_col] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__device__ void SplitKernelDetail(const T* input_data,
                                  const int64_t in_row,
                                  const int64_t in_col,
                                  const int64_t fixed_out_col,
                                  T** outputs_data) {
  CUDA_KERNEL_LOOP_TYPE(tid_x, in_col, int64_t) {
    int64_t split = tid_x / fixed_out_col;
    int64_t in_offset = tid_x - split * fixed_out_col;
    T* output_ptr = outputs_data[split];
    if (output_ptr != nullptr) {
      int64_t tid_y = blockIdx.y * blockDim.y + threadIdx.y;
      for (; tid_y < in_row; tid_y += blockDim.y * gridDim.y)
        output_ptr[tid_y * fixed_out_col + in_offset] =
            input_data[tid_y * in_col + tid_x];
    }
  }
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T** outputs_data) {
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1) {
  T* outputs_data[2];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1,
                             T* outputs_addr2) {
  T* outputs_data[3];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  outputs_data[2] = outputs_addr2;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

template <typename T>
__global__ void SplitKernel_(const T* input_data,
                             const int64_t in_row,
                             const int64_t in_col,
                             const int64_t fixed_out_col,
                             T* outputs_addr0,
                             T* outputs_addr1,
                             T* outputs_addr2,
                             T* outputs_addr3) {
  T* outputs_data[4];
  outputs_data[0] = outputs_addr0;
  outputs_data[1] = outputs_addr1;
  outputs_data[2] = outputs_addr2;
  outputs_data[3] = outputs_addr3;
  SplitKernelDetail<T>(input_data, in_row, in_col, fixed_out_col, outputs_data);
}

static inline void GetBlockDims(const phi::GPUContext& context,
                                int64_t num_rows,
                                int64_t num_cols,
                                dim3* block_dims,
                                dim3* grid_dims) {
  // Set the thread block and grid according to CurrentDeviceId
  const int kThreadsPerBlock = 1024;
  int block_cols = kThreadsPerBlock;
  if (num_cols < kThreadsPerBlock) {  // block_cols is aligned by 32.
    block_cols = ((num_cols + 31) >> 5) << 5;
  }
  int block_rows = kThreadsPerBlock / block_cols;
  *block_dims = dim3(block_cols, block_rows, 1);

  int max_threads = context.GetMaxPhysicalThreadCount();
  int64_t max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

  int grid_cols =
      std::min((num_cols + block_cols - 1) / block_cols, max_blocks);
  int grid_rows = std::min(max_blocks / grid_cols,
                           std::max(num_rows / block_rows, (int64_t)1));
  *grid_dims = dim3(grid_cols, grid_rows, 1);
}

/*
 * All tensors' dimension should be the same and the values of
 * each dimension must be the same, except the axis dimension.
 */
template <typename T, typename IndexT>
void ConcatFunctorWithIndexType(const phi::GPUContext& context,
                                const std::vector<phi::DenseTensor>& input,
                                int axis,
                                phi::DenseTensor* output) {
  // TODO(zcd): Add input data validity checking
  IndexT in_num = input.size();
  IndexT in_row = 1;
  auto dim_0 = input[0].dims();
  for (int i = 0; i < axis; ++i) {
    in_row *= dim_0[i];
  }
  IndexT in_col = input[0].numel() / in_row;
  IndexT out_row = in_row, out_col = 0;

  IndexT inputs_col_num = in_num + 1;
  std::vector<const T*> inputs_data_vec(in_num, nullptr);
  std::vector<IndexT> inputs_col_vec(inputs_col_num, 0);
  const T** inputs_data = inputs_data_vec.data();
  IndexT* inputs_col = inputs_col_vec.data();
#ifdef PADDLE_WITH_HIP
  // TODO(chentianyu03): try to find a method to remove the Alloc function
  paddle::memory::AllocationPtr data_alloc = paddle::memory::Alloc(
      paddle::platform::CUDAPinnedPlace(), in_num * sizeof(T*));
  inputs_data = reinterpret_cast<const T**>(data_alloc->ptr());
  paddle::memory::AllocationPtr col_alloc = paddle::memory::Alloc(
      paddle::platform::CUDAPinnedPlace(), inputs_col_num * sizeof(IndexT));
  inputs_col = reinterpret_cast<IndexT*>(col_alloc->ptr());
#endif

  bool has_same_shape = true;
  for (int i = 0; i < in_num; ++i) {
    IndexT t_cols = input[i].numel() / in_row;
    if (has_same_shape) {
      has_same_shape &= (t_cols == in_col);
    }
    out_col += t_cols;
    inputs_col[i + 1] = out_col;
  }
  dim3 block_dims;
  dim3 grid_dims;
  GetBlockDims(context, out_row, out_col, &block_dims, &grid_dims);
  IndexT limit_num = has_same_shape ? in_num : inputs_col_num;

#define IMPL_CONCATE_CUDA_KERNEL_HELPER(func_impl, ...) \
  func_impl(4, ##__VA_ARGS__);                          \
  func_impl(8, ##__VA_ARGS__);                          \
  func_impl(16, ##__VA_ARGS__);                         \
  func_impl(32, ##__VA_ARGS__);                         \
  func_impl(64, ##__VA_ARGS__);                         \
  func_impl(128, ##__VA_ARGS__);

  if (has_same_shape) {
#define IMPL_CONCAT_CUDA_KERNEL_CASE(size_, ...)                     \
  case size_: {                                                      \
    PointerWrapper<T, size_> ptr_array(context, input, inputs_data); \
    __VA_ARGS__;                                                     \
  } break;

    switch (phi::backends::gpu::RoundToNextHighPowOfTwo(limit_num, 4)) {
      IMPL_CONCATE_CUDA_KERNEL_HELPER(
          IMPL_CONCAT_CUDA_KERNEL_CASE,
          ConcatTensorWithSameShape<T, IndexT, decltype(ptr_array)>
          <<<grid_dims, block_dims, 0, context.stream()>>>(
              ptr_array, in_col, out_row, out_col, output->data<T>()));
      default: {
        PointerWrapper<T, 0> ptr_array(context, input, inputs_data);
        ConcatTensorWithSameShape<T, IndexT, decltype(ptr_array)>
            <<<grid_dims, block_dims, 0, context.stream()>>>(
                ptr_array, in_col, out_row, out_col, output->data<T>());
      }
    }
#undef IMPL_CONCAT_CUDA_KERNEL_CASE
  } else {
#define IMPL_COMPLEX_CONCAT_CUDA_KERNEL_CASE(size_, ...)          \
  case size_: {                                                   \
    PointerAndColWrapper<T, IndexT, size_> ptr_col_array(         \
        context, input, inputs_col_num, inputs_data, inputs_col); \
    __VA_ARGS__;                                                  \
  } break;

    switch (phi::backends::gpu::RoundToNextHighPowOfTwo(limit_num, 4)) {
      IMPL_CONCATE_CUDA_KERNEL_HELPER(
          IMPL_COMPLEX_CONCAT_CUDA_KERNEL_CASE,
          ConcatTensorWithDifferentShape<T, IndexT, decltype(ptr_col_array)>
          <<<grid_dims, block_dims, 0, context.stream()>>>(ptr_col_array,
                                                           inputs_col_num,
                                                           (out_row),
                                                           (out_col),
                                                           output->data<T>()));
      default: {
        PointerAndColWrapper<T, IndexT, 0> ptr_col_array(
            context, input, inputs_col_num, inputs_data, inputs_col);
        ConcatTensorWithDifferentShape<T, IndexT, decltype(ptr_col_array)>
            <<<grid_dims, block_dims, 0, context.stream()>>>(ptr_col_array,
                                                             inputs_col_num,
                                                             (out_row),
                                                             (out_col),
                                                             output->data<T>());
      }
    }
#undef IMPL_COMPLEX_CONCAT_CUDA_KERNEL_CASE
  }
#undef IMPL_CONCATE_CUDA_KERNEL_HELPER

#ifdef PADDLE_WITH_HIP
  // Prevent pinned memory from being covered and release the memory after
  // kernel launch of the stream is executed (reapply pinned memory next time)
  auto* data_alloc_released = data_alloc.release();
  auto* col_alloc_released = col_alloc.release();
  context.AddStreamCallback([data_alloc_released, col_alloc_released] {
    VLOG(4) << "Delete cuda pinned at " << data_alloc_released;
    VLOG(4) << "Delete cuda pinned at " << col_alloc_released;
    paddle::memory::allocation::Allocator::AllocationDeleter(
        data_alloc_released);
    paddle::memory::allocation::Allocator::AllocationDeleter(
        col_alloc_released);
  });
#endif
}

template <typename T>
struct ConcatFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const std::vector<phi::DenseTensor>& input,
                  int axis,
                  phi::DenseTensor* output) {
    if (output->numel() < std::numeric_limits<int32_t>::max()) {
      ConcatFunctorWithIndexType<T, int32_t>(context, input, axis, output);
    } else {
      ConcatFunctorWithIndexType<T, int64_t>(context, input, axis, output);
    }
  }
};

template <typename T>
class SplitFunctor<phi::GPUContext, T> {
 public:
  void operator()(const phi::GPUContext& context,
                  const phi::DenseTensor& input,
                  const std::vector<const phi::DenseTensor*>& ref_inputs,
                  int axis,
                  std::vector<phi::DenseTensor*>* outputs) {
    // NOTE(zhiqiu): split a tensor of shape [0,3,4] at axis=1, result in 3
    // tensors of shape [0,1,4]
    if (input.numel() == 0) {
      return;
    }

    // TODO(zcd): Add input data validity checking
    int o_num = outputs->size();
    int64_t out_row = 1;
    auto dim_0 = ref_inputs[0]->dims();
    for (int i = 0; i < axis; ++i) {
      out_row *= dim_0[i];
    }

    int64_t out0_col = ref_inputs[0]->numel() / out_row;
    int64_t in_col = 0, in_row = out_row;
    bool has_same_shape = true;

    int outputs_cols_num = o_num + 1;
    std::vector<T*> outputs_data_vec(o_num);
    std::vector<int64_t> outputs_cols_vec(outputs_cols_num);
    T** outputs_data = outputs_data_vec.data();
    int64_t* outputs_cols = outputs_cols_vec.data();

// There are some differences between hip runtime and NV runtime.
// In NV, when the pageable memory data less than 64K is transferred from
// hosttodevice, it will be automatically asynchronous.
// However, only pinned memory in hip can copy asynchronously
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#concurrent-execution-host-device
// 3.2.6.1. Concurrent Execution between Host and Device
// Memory copies from host to device of a memory block of 64 KB or less
#ifdef PADDLE_WITH_HIP
    paddle::memory::AllocationPtr data_alloc, cols_alloc;
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    data_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       o_num * sizeof(T*));
    outputs_data = reinterpret_cast<T**>(data_alloc->ptr());
    // TODO(chentianyu03): try to find a method to remove the Alloc function
    cols_alloc = paddle::memory::Alloc(paddle::platform::CUDAPinnedPlace(),
                                       (outputs_cols_num) * sizeof(int64_t));
    outputs_cols = reinterpret_cast<int64_t*>(cols_alloc->ptr());
#endif

    outputs_cols[0] = 0;
    for (int i = 0; i < o_num; ++i) {
      int64_t t_col = ref_inputs.at(i)->numel() / out_row;
      if (has_same_shape) {
        if (t_col != out0_col) has_same_shape = false;
      }
      in_col += t_col;
      outputs_cols[i + 1] = in_col;
      if (outputs->at(i) != nullptr) {
        outputs_data[i] = outputs->at(i)->data<T>();
      } else {
        outputs_data[i] = nullptr;
      }
    }

    dim3 block_dims;
    dim3 grid_dims;
    GetBlockDims(context, out_row, in_col, &block_dims, &grid_dims);

    paddle::memory::allocation::AllocationPtr tmp_dev_outs_data;
    T** dev_out_gpu_data = nullptr;
    if (!has_same_shape || o_num < 2 || o_num > 4) {
      // TODO(chentianyu03): try to find a method to remove the Alloc function
      tmp_dev_outs_data = paddle::memory::Alloc(
          context.GetPlace(),
          o_num * sizeof(T*),
          phi::Stream(reinterpret_cast<phi::StreamId>(context.stream())));
      auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
          outputs_data, o_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_outs_data->ptr(),
                           phi::CPUPlace(),
                           restored,
                           o_num * sizeof(T*),
                           context.stream());
      dev_out_gpu_data = reinterpret_cast<T**>(tmp_dev_outs_data->ptr());
    }

    if (has_same_shape) {
      if (o_num == 2) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1]);
      } else if (o_num == 3) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1],
            outputs_data[2]);
      } else if (o_num == 4) {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(),
            in_row,
            in_col,
            out0_col,
            outputs_data[0],
            outputs_data[1],
            outputs_data[2],
            outputs_data[3]);
      } else {
        SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
            input.data<T>(), in_row, in_col, out0_col, dev_out_gpu_data);
      }
    } else {
      auto tmp_dev_ins_col_data =
          // TODO(chentianyu03): try to find a method to remove the Alloc
          // function
          paddle::memory::Alloc(
              context.GetPlace(),
              outputs_cols_num * sizeof(int64_t),
              phi::Stream(reinterpret_cast<phi::StreamId>(context.stream())));
      auto* restored = phi::backends::gpu::RestoreHostMemIfCapturingCUDAGraph(
          outputs_cols, outputs_cols_num);
      paddle::memory::Copy(context.GetPlace(),
                           tmp_dev_ins_col_data->ptr(),
                           phi::CPUPlace(),
                           restored,
                           outputs_cols_num * sizeof(int64_t),
                           context.stream());
      int64_t* dev_outs_col_data =
          reinterpret_cast<int64_t*>(tmp_dev_ins_col_data->ptr());

      SplitKernel_<<<grid_dims, block_dims, 0, context.stream()>>>(
          input.data<T>(),
          in_row,
          in_col,
          dev_outs_col_data,
          static_cast<int>(outputs_cols_num),
          dev_out_gpu_data);
    }

#ifdef PADDLE_WITH_HIP
    // Prevent the pinned memory value from being covered and release the memory
    // after the launch kernel of the stream is executed (reapply pinned memory
    // next time)
    auto* data_alloc_released = data_alloc.release();
    auto* cols_alloc_released = cols_alloc.release();
    context.AddStreamCallback([data_alloc_released, cols_alloc_released] {
      paddle::memory::allocation::Allocator::AllocationDeleter(
          data_alloc_released);
      paddle::memory::allocation::Allocator::AllocationDeleter(
          cols_alloc_released);
    });
#endif
  }
};

#define DEFINE_FUNCTOR(type)                           \
  template class ConcatFunctor<phi::GPUContext, type>; \
  template class SplitFunctor<phi::GPUContext, type>

FOR_ALL_TYPES(DEFINE_FUNCTOR);

}  // namespace funcs
}  // namespace phi
