#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>

#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/fc_functor.h"

namespace phi {
namespace funcs {

using float16 = phi::dtype::float16;

template <typename T>
struct FcTypeTraits;

template <>
struct FcTypeTraits<float> {
  typedef float4 Type;
};

template <>
struct FcTypeTraits<double> {
  typedef double4 Type;
};

#if defined(PADDLE_WITH_CUDA)
#include <hip/hip_fp16.h>

template <>
struct FcTypeTraits<float16> {
  typedef half2 Type;
};
#else
struct float16_4 {
  float16 x, y, z, w;
};

template <>
struct FcTypeTraits<float16> {
  typedef float16_4 Type;
};
#endif

template <typename T, bool DoRelu>
__global__ void bias_relu_v4(const int num, const T* bias, T* data, int K) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int bias_idx = tid % K;
    const T bias_ptr = bias[bias_idx];
    const T in_ptr = data[tid];
    T packed_val;
    packed_val.x = in_ptr.x + bias_ptr.x;
    packed_val.y = in_ptr.y + bias_ptr.y;
    packed_val.z = in_ptr.z + bias_ptr.z;
    packed_val.w = in_ptr.w + bias_ptr.w;
    if (DoRelu) {
      packed_val.x = fmaxf(0.f, packed_val.x);
      packed_val.y = fmaxf(0.f, packed_val.y);
      packed_val.z = fmaxf(0.f, packed_val.z);
      packed_val.w = fmaxf(0.f, packed_val.w);
    }
    data[tid] = packed_val;
  }
}

template <typename T, bool DoRelu, int BlockDim>
__global__ void InplaceAddReluKernel(const int N, const T* bias, T* data) {
  int offset = blockIdx.x * N;

  for (int i = threadIdx.x; i < N; i += BlockDim) {
    T temp;
#if defined(__HIPCC__) || __CUDA_ARCH__ >= 350
    temp = __ldg(data + offset + i) + __ldg(bias + i);
#else
    temp = data[offset + i] + bias[i];
#endif
    if (DoRelu) {
      data[offset + i] = static_cast<int>(temp > 0) * temp;
    } else {
      data[offset + i] = temp;
    }
  }
}

template <typename T>
void AddReluKernel(
    gpuStream_t stream, const int M, const int N, T* Y, const T* B, bool relu) {
  if (N % 4 == 0) {
    const int threads = 256;
    const int num = M * N / 4;
    const int blocks = (num + threads - 1) / threads;
    typedef typename FcTypeTraits<T>::Type trans_type;
    auto* bias_ptr_v4 = reinterpret_cast<const trans_type*>(B);
    auto* data_ptr_v4 = reinterpret_cast<trans_type*>(Y);
    if (relu) {
      bias_relu_v4<trans_type, true><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    } else {
      bias_relu_v4<trans_type, false><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    }
  } else {
    const int threads = 256;
    const int blocks = M;

    if (relu) {
      InplaceAddReluKernel<T, true, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    } else {
      InplaceAddReluKernel<T, false, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    }
  }
}

#if defined(PADDLE_WITH_CUDA)
template <bool DoRelu>
__global__ void bias_relu_v2(const int num,
                             const half2* bias,
                             half2* data,
                             int K) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < num) {
    int bias_idx = tid % K;
    const half2 bias_ptr = bias[bias_idx];
    const half2 in_ptr = data[tid];
    half2 packed_val;
#if __CUDA_ARCH__ >= 530
    packed_val = __hadd2(bias_ptr, in_ptr);
#else
    packed_val.x = __hadd(bias_ptr.x, in_ptr.x);
    packed_val.y = __hadd(bias_ptr.y, in_ptr.y);
#endif
    if (DoRelu) {
#if __CUDA_ARCH__ >= 800
      packed_val = __hmax2(__half2(0, 0), packed_val);
#elif __CUDA_ARCH__ >= 530
      packed_val = __hmul2(__hgt2(packed_val, __half2(0, 0)), packed_val);
#else
      packed_val.x = static_cast<int>(static_cast<float>(packed_val.x) > 0) *
                     static_cast<float>(packed_val.x);
      packed_val.y = static_cast<int>(static_cast<float>(packed_val.y) > 0) *
                     static_cast<float>(packed_val.y);
#endif
    }
    data[tid] = packed_val;
  }
}

template <bool DoRelu, int BlockDim>
__global__ void InplaceAddReluKernel(const int N,
                                     const half* bias,
                                     half* data) {
  int offset = blockIdx.x * N;
  for (int i = threadIdx.x; i < N; i += BlockDim) {
    half temp;
#if defined(__HIPCC__) || __CUDA_ARCH__ >= 350
    temp = __hadd(__ldg(data + offset + i), __ldg(bias + i));
#else
    temp = __hadd(data[offset + i], bias[i]);
#endif
    if (DoRelu) {
#if __CUDA_ARCH__ >= 800
      data[offset + i] = __hmax(0, temp);
#elif __CUDA_ARCH__ >= 530
      data[offset + i] = __hmul(__hgt(temp, 0), temp);
#else
      data[offset + i] = static_cast<int>(static_cast<float>(temp) > 0) *
                         static_cast<float>(temp);
#endif
    } else {
      data[offset + i] = temp;
    }
  }
}

template <>
void AddReluKernel(hipStream_t stream,
                   const int M,
                   const int N,
                   float16* Y,
                   const float16* B,
                   bool relu) {
  if (N % 2 == 0) {
    const int threads = 256;
    const int num = M * N / 2;
    const int blocks = (num + threads - 1) / threads;
    typedef typename FcTypeTraits<float16>::Type trans_type;
    auto* bias_ptr_v2 = reinterpret_cast<const trans_type*>(B);
    auto* data_ptr_v2 = reinterpret_cast<trans_type*>(Y);
    if (relu) {
      bias_relu_v2<true><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v2, data_ptr_v2, N / 2);
    } else {
      bias_relu_v2<false><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v2, data_ptr_v2, N / 2);
    }
  } else {
    const int threads = 256;
    const int blocks = M;
    auto* halfB = reinterpret_cast<const half*>(B);
    auto* halfY = reinterpret_cast<half*>(Y);
    if (relu) {
      InplaceAddReluKernel<true, threads>
          <<<blocks, threads, 0, stream>>>(N, halfB, halfY);
    } else {
      InplaceAddReluKernel<false, threads>
          <<<blocks, threads, 0, stream>>>(N, halfB, halfY);
    }
  }
}
#else
template <bool DoRelu, int BlockDim>
__global__ void InplaceAddReluKernel(const int N,
                                     const float16* bias,
                                     float16* data) {
  int offset = blockIdx.x * N;
  for (int i = threadIdx.x; i < N; i += BlockDim) {
    float16 temp;
    temp = data[offset + i] + bias[i];
    if (DoRelu) {
      data[offset + i] = fmaxf(0.f, temp);
    } else {
      data[offset + i] = temp;
    }
  }
}

template <>
void AddReluKernel(gpuStream_t stream,
                   const int M,
                   const int N,
                   float16* Y,
                   const float16* B,
                   bool relu) {
  if (N % 4 == 0) {
    const int threads = 256;
    const int num = M * N / 4;
    const int blocks = (num + threads - 1) / threads;
    typedef typename FcTypeTraits<float16>::Type trans_type;
    auto* bias_ptr_v4 = reinterpret_cast<const trans_type*>(B);
    auto* data_ptr_v4 = reinterpret_cast<trans_type*>(Y);
    if (relu) {
      bias_relu_v4<trans_type, true><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    } else {
      bias_relu_v4<trans_type, false><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    }
  } else {
    const int threads = 256;
    const int blocks = M;

    if (relu) {
      InplaceAddReluKernel<true, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    } else {
      InplaceAddReluKernel<false, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    }
  }
}
#endif

template <typename DeviceContext, typename T>
void FCFunctor<DeviceContext, T>::operator()(const DeviceContext& context,
                                             const int M,
                                             const int N,
                                             const int K,
                                             const T* X,
                                             const T* W,
                                             T* Y,
                                             const T* B,
                                             bool relu,
                                             bool padding_weights) {
  PADDLE_ENFORCE_EQ(padding_weights,
                    false,
                    errors::PermissionDenied(
                        "Weight padding in fc can not be used in GPU scope."));
  auto blas = phi::funcs::GetBlas<DeviceContext, T>(context);
  blas.GEMM(CblasNoTrans,
            CblasNoTrans,
            M,
            N,
            K,
            static_cast<T>(1.0),
            X,
            W,
            static_cast<T>(0.0),
            Y);
  if (B == NULL) {
    return;
  }

  // M * N
  AddReluKernel(context.stream(), M, N, Y, B, relu);
}

template class FCFunctor<GPUContext, float16>;
template class FCFunctor<GPUContext, float>;
template class FCFunctor<GPUContext, double>;

}  // namespace funcs
}  // namespace phi
