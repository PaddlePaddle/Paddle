#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_MUSA
#include <musa.h>
#include <musa_runtime.h>
#endif

#ifdef PADDLE_WITH_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>  // NOLINT
#endif
#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/phi/kernels/funcs/emb_eltwise_layer_norm_functor.h"

#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"

namespace phi {
namespace funcs {

template <typename T>
__device__ inline T rsqrt(const T& x);

template <>
__device__ inline float rsqrt(const float& x) {
  return rsqrtf(x);
}

template <typename T>
__device__ __forceinline__ T local_rsqrt(T num) {
  return rsqrt(static_cast<float>(num));
}
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
__device__ __forceinline__ half local_rsqrt(half num) { return hrsqrt(num); }
#endif

template <typename T, int TPB>
__device__ inline void LayerNorm(const phi::funcs::kvp<T>& thread_data,
                                 const int ld,
                                 const int offset,
                                 const T* bias,
                                 const T* scale,
                                 T* output,
                                 T eps) {
  using BlockReduce = hipcub::BlockReduce<phi::funcs::kvp<T>, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T mu;      // mean
  __shared__ T rsigma;  // 1 / std.dev.

  const auto sum_kv = BlockReduce(temp_storage).Reduce(thread_data, hipcub::Sum());

  if (threadIdx.x == 0) {
    mu = sum_kv.key;
    rsigma = local_rsqrt(sum_kv.value - mu * mu + eps);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    const T val = output[idx];
    const T g(scale[i]);
    const T b(bias[i]);
    output[idx] = g * (val - mu) * rsigma + b;
  }
}

template <typename T, unsigned TPB>
__global__ void EmbEltwiseLayernormKernel(int hidden,
                                          const int64_t* ids,
                                          const T* scale,
                                          const T* bias,
                                          const int64_t* embs,
                                          T* output,
                                          T eps,
                                          int input_num) {
  hipcub::Sum pair_sum;
  // blockIdx.x: position in the sequence
  // blockIdx.y: batch
  // gridDim.x: Seq
  // gridDim.y: Batch

  extern __shared__ int64_t array_id[];

  const T rhidden = T(1.f) / T(hidden);
  const int64_t seq_pos = blockIdx.y + blockIdx.x * gridDim.y;
  if (threadIdx.x == 0) {
    for (int i = 0; i < input_num; ++i) {
      const int64_t* ids_p = reinterpret_cast<const int64_t*>(ids[i]);
      array_id[i] = ids_p[seq_pos];
    }
  }
  __syncthreads();

  const int64_t out_offset = seq_pos * hidden;

  phi::funcs::kvp<T> thread_data(0, 0);

#pragma unroll
  for (int it = threadIdx.x; it < hidden; it += TPB) {
    T val = 0;
    for (int i = 0; i < input_num; ++i) {
      val += reinterpret_cast<const T*>(embs[i])[array_id[i] * hidden + it];
    }

    output[out_offset + it] = val;
    const T rhiddenval = rhidden * val;
    thread_data =
        pair_sum(thread_data, phi::funcs::kvp<T>(rhiddenval, rhiddenval * val));
  }
  LayerNorm<T, TPB>(thread_data, hidden, out_offset, bias, scale, output, eps);
}

// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#ifdef __HIPCC__  // @{ Half kernel: EmbEltwiseLayernormKernel
template <>
__global__ void EmbEltwiseLayernormKernel<half, 256>(int hidden,
                                                     const int64_t* ids,
                                                     const half* scale,
                                                     const half* bias,
                                                     const int64_t* embs,
                                                     half* output,
                                                     half eps,
                                                     int input_num) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  hipcub::Sum pair_sum;
  // blockIdx.x: position in the sequence
  // blockIdx.y: batch
  // gridDim.x: Seq
  // gridDim.y: Batch

  extern __shared__ int64_t array_id[];

  const half rhidden = half(1.f) / half(hidden);
  const int64_t seq_pos = blockIdx.y + blockIdx.x * gridDim.y;
  if (threadIdx.x == 0) {
    for (int i = 0; i < input_num; ++i) {
      const int64_t* ids_p = reinterpret_cast<const int64_t*>(ids[i]);
      array_id[i] = ids_p[seq_pos];
    }
  }
  __syncthreads();

  const int64_t out_offset = seq_pos * hidden;

  phi::funcs::kvp<half> thread_data(0, 0);

#pragma unroll
  for (int it = threadIdx.x; it < hidden; it += 256) {
    half val = 0;
    for (int i = 0; i < input_num; ++i) {
      val += reinterpret_cast<const half*>(embs[i])[array_id[i] * hidden + it];
    }

    output[out_offset + it] = val;
    const half rhiddenval = rhidden * val;
    thread_data = pair_sum(thread_data,
                           phi::funcs::kvp<half>(rhiddenval, rhiddenval * val));
  }
  LayerNorm<half, 256>(
      thread_data, hidden, out_offset, bias, scale, output, eps);
#endif
}
#endif  // @} End Half kernel: EmbEltwiseLayernormKernel

template <typename T>
void EmbEltwiseLayerNormFunctor<T>::operator()(int batch,
                                               int seq_len,
                                               int hidden,
                                               const int64_t* ids,
                                               const T* scale,
                                               const T* bias,
                                               const int64_t* embs,
                                               T* output,
                                               float eps,
                                               int input_num,
                                               gpuStream_t stream) {
  const unsigned tpb = 256;
  const dim3 grid(seq_len, batch, 1);
  const dim3 block(tpb, 1, 1);
  int shared_bytes = input_num * sizeof(int64_t);
  EmbEltwiseLayernormKernel<T, tpb><<<grid, block, shared_bytes, stream>>>(
      hidden, ids, scale, bias, embs, output, eps, input_num);
}

template class EmbEltwiseLayerNormFunctor<float>;

// device function 'operator()' is not supportted until cuda 10.0
// HIP defined __HIP_NO_HALF_CONVERSIONS__ in hip.cmake
#if defined(PADDLE_WITH_CUDA) && CUDA_VERSION >= 10000
template class EmbEltwiseLayerNormFunctor<half>;
#endif

}  // namespace funcs
}  // namespace phi
