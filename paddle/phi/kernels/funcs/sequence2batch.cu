#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include "paddle/phi/kernels/funcs/sequence2batch.h"

namespace phi {
namespace funcs {

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void CopyMatrixRowsKernel(const T* src,
                                     T* dst,
                                     const size_t* index,
                                     int64_t height,
                                     int64_t width,
                                     bool is_src_index) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id = blockIdx.x + idy * GridDimX;
  while (id < height) {
    int src_idx = is_src_index ? index[id] : id;
    int dst_idx = is_src_index ? id : index[id];
    const T* src_data = src + src_idx * width;
    T* dst_data = dst + dst_idx * width;
    for (int i = idx; i < width; i += BlockDimX) {
      dst_data[i] = src_data[i];
    }
    id += BlockDimY * GridDimX;
  }
}

template <typename T>
class CopyMatrixRowsFunctor<paddle::platform::CUDADeviceContext, T> {
 public:
  void operator()(const paddle::platform::CUDADeviceContext& context,
                  const paddle::framework::Tensor& src,
                  paddle::framework::Vector<size_t> index_lod,
                  paddle::framework::Tensor* dst,
                  bool is_src_index) {
    auto src_dims = src.dims();
    auto dst_dims = dst->dims();
    PADDLE_ENFORCE_EQ(src_dims.size(),
                      2,
                      phi::errors::InvalidArgument(
                          "The source tensor must be a matrix with rank 2, but "
                          "got the source tensor rank is %lu. "
                          "Please check the rank of the source tensor",
                          src_dims.size()));
    PADDLE_ENFORCE_EQ(dst_dims.size(),
                      2,
                      phi::errors::InvalidArgument(
                          "The destination tensor must be a matrix with rank, "
                          "but got the destination tensor rank is %lu. "
                          "Please check the rank of the destination tensor",
                          dst_dims.size()));
    PADDLE_ENFORCE_EQ(
        src_dims[1],
        dst_dims[1],
        phi::errors::InvalidArgument(
            "The width of the source tensor and the destination tensor must be "
            "same. But got %lu != %lu.Please check the rank of the source "
            "tensor",
            src_dims.size(),
            dst_dims.size()));
    auto height = dst_dims[0];
    auto width = dst_dims[1];
    auto* src_data = src.data<T>();
    auto* dst_data = dst->data<T>();

    dim3 threads(128, 8);
    dim3 grid(8, 1);
    auto stream = context.stream();
    paddle::framework::MixVector<size_t> mix_index_lod(&index_lod);
    CopyMatrixRowsKernel<T, 128, 8, 8><<<grid, threads, 0, stream>>>(
        src_data,
        dst_data,
        mix_index_lod.CUDAData(context.GetPlace()),
        height,
        width,
        is_src_index);
  }
};

template class CopyMatrixRowsFunctor<paddle::platform::CUDADeviceContext,
                                     float>;
template class CopyMatrixRowsFunctor<paddle::platform::CUDADeviceContext,
                                     double>;

template class LoDTensor2BatchFunctor<paddle::platform::CUDADeviceContext,
                                      float>;
template class LoDTensor2BatchFunctor<paddle::platform::CUDADeviceContext,
                                      double>;
template class Batch2LoDTensorFunctor<paddle::platform::CUDADeviceContext,
                                      float>;
template class Batch2LoDTensorFunctor<paddle::platform::CUDADeviceContext,
                                      double>;

}  // namespace funcs
}  // namespace phi
