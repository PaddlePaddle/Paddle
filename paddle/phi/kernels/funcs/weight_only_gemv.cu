#include "hip/hip_runtime.h"
/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/funcs/weight_only_gemv.h"

#include <assert.h>
#include <stdint.h>
#include <cmath>
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/datatype_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"

// #define _DEBUG_WEIGHT_ONLY_GEMV

namespace phi {

namespace {

#ifdef PADDLE_WITH_CUDA
constexpr int kWarpSize = 32;
constexpr int kPerBlockWarpNum = 8;

/////////////////////////////////////////////////////////////////////

template <typename T>
struct CUDA_HALF_2_TYPE_TARIS {};

template <>
struct CUDA_HALF_2_TYPE_TARIS<half> {
  using type = half2;
};

#ifdef PADDLE_CUDA_BF16
template <>
struct CUDA_HALF_2_TYPE_TARIS<__hip_bfloat16> {
  using type = __hip_bfloat162;
};
#endif

template <typename T>
__device__ inline void fast_cvt_4_packed_signed_i8s_to_2_half2s(
    T halves[4], int8_t signed_chars[4]) {
  assert(false);
}

// Specialization for fast cast from FP16 -> int8
template <>
__device__ inline void fast_cvt_4_packed_signed_i8s_to_2_half2s(
    half halves[4], int8_t signed_chars[4]) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  uint32_t* h = reinterpret_cast<uint32_t*>(halves);
  uint32_t i8s = *reinterpret_cast<uint32_t*>(signed_chars);

  static constexpr uint32_t mask_for_elt_01 = 0x5150;
  static constexpr uint32_t mask_for_elt_23 = 0x5352;
  static constexpr uint32_t start_byte_for_fp16 = 0x64646464;
  asm volatile("prmt.b32 %0,%1,%2,%3;\n"
               : "=r"(h[0])
               : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
  asm volatile("prmt.b32 %0,%1,%2,%3;\n"
               : "=r"(h[1])
               : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));

  static constexpr uint32_t I8s_TO_F16s_MAGIC_NUM = 0x64806480;
  asm volatile("sub.f16x2 %0, %1, %2;\n"
               : "=r"(h[0])
               : "r"(h[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
  asm volatile("sub.f16x2 %0, %1, %2;\n"
               : "=r"(h[1])
               : "r"(h[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
#endif
}

// Specialization for fast cast from BF16 -> int8
#ifdef PADDLE_CUDA_BF16
template <>
__device__ inline void fast_cvt_4_packed_signed_i8s_to_2_half2s(
    __hip_bfloat16 halves[4], int8_t signed_chars[4]) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
  uint32_t* bf16_result_ptr = reinterpret_cast<uint32_t*>(halves);
  uint32_t i8s = *reinterpret_cast<uint32_t*>(signed_chars);

  static constexpr uint32_t fp32_base = 0x4B000000;
  float fp32_intermediates[4];

  // Construct FP32s, bfloat does not have enough mantissa for IADD trick
  uint32_t* fp32_intermediates_casted =
      reinterpret_cast<uint32_t*>(fp32_intermediates);
  fp32_intermediates_casted[0] = __byte_perm(i8s, fp32_base, 0x7650);
  fp32_intermediates_casted[1] = __byte_perm(i8s, fp32_base, 0x7651);
  fp32_intermediates_casted[2] = __byte_perm(i8s, fp32_base, 0x7652);
  fp32_intermediates_casted[3] = __byte_perm(i8s, fp32_base, 0x7653);

// Subtract out fp32_base + 128 to make the unsigned integer signed.
#pragma unroll
  for (int ii = 0; ii < 4; ++ii) {
    fp32_intermediates[ii] -= 8388736.f;
  }

// Truncate the fp32 representation and pack up as bfloat16s.
#pragma unroll
  for (int ii = 0; ii < 2; ++ii) {
    bf16_result_ptr[ii] = __byte_perm(fp32_intermediates_casted[2 * ii + 0],
                                      fp32_intermediates_casted[2 * ii + 1],
                                      0x7632);
  }
#else
  // Disable this on architectures older than Ampere since they lack hardware
  // for bf16 mma. If one wishes to use HMMA on older hardware, they should
  // Convert directly to FP16 using FP16 converters.
  assert(false);
#endif
}
#endif

/* Gelu Activation */

__forceinline__ __device__ float copysignf_pos(float a, float b) {
  float r;
  r = __int_as_float(__float_as_int(a) | (__float_as_int(b) & 0x80000000));
  return r;
}

__inline__ __device__ float tanh_opt(float x) {
#if (__CUDA_ARCH__ >= 750 && CUDART_VERSION >= 11000)
  float r;
  asm("tanh.approx.f32 %0,%1; \n\t" : "=f"(r) : "f"(x));
  return r;
#else
  const float exp_val = -1.f * fabs(2 * x);
  return copysignf_pos((1.0f - __expf(exp_val)) / (__expf(exp_val) + 1.0f), x);
#endif
}

template <typename T, bool EnableFastGelu>
struct GeluActivation {
  using return_type = T;
  static __device__ __forceinline__ T apply(const T& val) {
    if (!EnableFastGelu) return val;
    const float cdf =
        0.5f * (1.0f + tanh_opt((0.7978845608028654f *
                                 (val + 0.044715f * val * val * val))));
    return val * cdf;
  }
};

template <typename T>
struct ConvertFloatFunc {
  ConvertFloatFunc() {}
  static __device__ __forceinline__ float apply(const T& val) {
    assert(false);
    return 0.0f;
  }
};

template <>
struct ConvertFloatFunc<half> {
  static __device__ __forceinline__ float apply(const half& val) {
    return __half2float(val);
  }
};

#ifdef PADDLE_CUDA_BF16
template <>
struct ConvertFloatFunc<__hip_bfloat16> {
  static __device__ __forceinline__ float apply(const __hip_bfloat16& val) {
    return __bfloat162float(val);
  }
};
#endif

template <typename T>
struct ConvertDstFunc_2 {
  static __device__ __forceinline__ T apply(const float& val) { assert(false); }
};

template <typename T>
struct ConvertDstFunc {
  static __device__ __forceinline__ T apply(const float& val) { assert(false); }
};

template <>
struct ConvertDstFunc<half> {
  static __device__ __forceinline__ half apply(const float& val) {
    return __float2half_rn(val);
  }
};

template <>
struct ConvertDstFunc<half2> {
  static __device__ __forceinline__ half2 apply(const float& val) {
    return __float2half2_rn(val);
  }
};

template <>
struct ConvertDstFunc_2<half2> {
  static __device__ __forceinline__ half2 apply(const half& val) {
    return __half2half2(val);
  }
};
#ifdef PADDLE_CUDA_BF16
template <>
struct ConvertDstFunc<__hip_bfloat16> {
  static __device__ __forceinline__ __hip_bfloat16 apply(const float& val) {
    return __float2bfloat16_rn(val);
  }
};

template <>
struct ConvertDstFunc<__hip_bfloat162> {
  static __device__ __forceinline__ __hip_bfloat162 apply(const float& val) {
    return __float2bfloat162_rn(val);
  }
};

template <>
struct ConvertDstFunc_2<__hip_bfloat162> {
  static __device__ __forceinline__ __hip_bfloat162
  apply(const __hip_bfloat16& val) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800
    __hip_bfloat162 val2;
    val2.x = val;
    val2.y = val;
    return val2;
#else
    return __bfloat162bfloat162(val);
#endif
  }
};
#endif

template <typename T>
struct HalfMul {
  static __device__ __forceinline__ T apply(const T& x, const T& y) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 530
    return __hmul(x, y);
#else
    float res = static_cast<float>(float16(x)) * static_cast<float>(float16(y));
    return float16(res).to_half();
#endif
  }
};

template <typename T>
struct HalfMulAdd {
  static __device__ __forceinline__ T apply(const T& x,
                                            const T& y,
                                            const T& z) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 530
    return __hfma2(x, y, z);
#else
    assert(0 && "HalfMulAdd cuda version error");
#endif
  }
};

#ifdef PADDLE_CUDA_BF16
template <>
struct HalfMul<__hip_bfloat16> {
  static __device__ __forceinline__ __hip_bfloat16
  apply(const __hip_bfloat16& x, const __hip_bfloat16& y) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    return __hmul(x, y);
#else
    return __float2bfloat16_rn(__bfloat162float(x) * __bfloat162float(y));
#endif
  }
};

template <>
struct HalfMulAdd<__hip_bfloat162> {
  static __device__ __forceinline__ __hip_bfloat162
  apply(const __hip_bfloat162& x,
        const __hip_bfloat162& y,
        const __hip_bfloat162& z) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
    return __hfma2(x, y, z);
#else
    assert(0 && "HalfMulAdd cuda version error");
#endif
  }
};
#endif

/*
Int8 Weightonly GEMV.
X: 1 x k
Weight(ColMajor): n x k
Each Warp Process: 1 x k matmul 1 x k
*/
template <typename T, bool Bias, bool Gelu>
__global__ void int8_weight_only_gemv(const T* input,
                                      const int8_t* weight,
                                      const T* scale_list,
                                      const T* bias,
                                      T* output,
                                      const int k,
                                      const int n) {
  constexpr int kWarpSize = 32;
  constexpr int kVecSize = 16;
  T vec_input[kVecSize];
  int8_t vec_weight[kVecSize];
  T vec_weight_f16[kVecSize];

  const int warp_id = threadIdx.x / kWarpSize;
  const int lane_id = threadIdx.x % kWarpSize;
  const int tile_id = blockIdx.x * blockDim.x / kWarpSize + warp_id;
  const int row_id = tile_id * 2 + ((lane_id % 8) > 3 ? 1 : 0);
  weight += tile_id * k * 2;

  float v = 0.f, scale = static_cast<float>(scale_list[row_id]), v_bias;

  if (Bias) {
    v_bias = ConvertFloatFunc<T>::apply(bias[row_id]);
  }

#pragma unroll
  for (int i = lane_id * kVecSize; i < k * 2; i += kVecSize * kWarpSize) {
    *reinterpret_cast<int4*>(vec_weight) =
        *reinterpret_cast<const int4*>(weight + i);  // NOLINT
    *reinterpret_cast<float4*>(vec_input) =          // NOLINT
        *reinterpret_cast<const float4*>(input + i / 128 * 64 +
                                         (i % 64));  // NOLINT
    *reinterpret_cast<float4*>(vec_input + 8) =      // NOLINT
        *reinterpret_cast<const float4*>(input + i / 128 * 64 + (i % 64) +
                                         8);  // NOLINT

#pragma unroll
    for (int p = 0; p < kVecSize; p += 4) {
      fast_cvt_4_packed_signed_i8s_to_2_half2s<T>(vec_weight_f16 + p,
                                                  vec_weight + p);
    }
#pragma unroll
    for (int p = 0; p < kVecSize; ++p) {
      v += ConvertFloatFunc<T>::apply(
          HalfMul<T>::apply(vec_input[p], vec_weight_f16[p / 8 + (p % 8) * 2]));
    }
  }
  // Do WarpReduceSum.
  v += __shfl_xor_sync(0xffffffff, v, 16);
  v += __shfl_xor_sync(0xffffffff, v, 8);
  v += __shfl_xor_sync(0xffffffff, v, 2);
  v += __shfl_xor_sync(0xffffffff, v, 1);
  if (lane_id == 0 || lane_id == 4) {
    if (Bias) {
      output[row_id] = ConvertDstFunc<T>::apply(
          GeluActivation<float, Gelu>::apply(v * scale + v_bias));
    } else {
      output[row_id] = ConvertDstFunc<T>::apply(
          GeluActivation<float, Gelu>::apply(v * scale));
    }
  }
}

enum class WeightOnlyQuantType { Int4b, Int8b };

enum class WeightOnlyType { PerChannel, GroupWise };

template <WeightOnlyQuantType QType>
struct WeightLayoutDetails;

template <>
struct WeightLayoutDetails<WeightOnlyQuantType::Int4b> {
  // Every four rows of the original weights are interleaved into a row with
  // stride of 64, so if each thread processes 32 elements(for int4, we can use
  // ldg.128 to load weights), then every group of two adjacent threads will
  // alternately process four different row weights for example every 256
  // consecutive int4 elements [256*i, 256*(i+1)-1] of row N under interleave
  // layout, the first 64 are from [64*i, 64*(i+1)-1] of row 4N before
  // interleaving, and the second 64 are from [64*i, 64*(i+1)-1] of row 4N+1
  // before interleaving, and so on. So if each thread loads 32 int4 elements,
  // then the elements of each 2 adjacent threads of each 8 consecutive threads
  // will come from row 4N ~ 4N+3 respectively before interleaving.
  static constexpr int kElemBits = 4;
  static constexpr int kInterleave = 4;
  static constexpr int kStride = 64;

  // The index remapping here is to counteracts the effect of
  // cutlass::permute_B_rows_for_mixed_gemm input 0 1 2 3 4 5 6 7 8 9 10 11 12
  // 13 14 15 ... 31 weight 0 1 8 9 16 17 24 25 2 3 10 11 18 19 26 27 4 5 12 13
  // 20 21 28 29 6 7 14 15 22 23 30 31
  static constexpr int kShuffleSize = 32;
  static constexpr int kShuffleBasicTile = 2;
  static constexpr int kShuffleContinous = 4;
  static constexpr int kShuffleStrided = 4;

  // The rearrangement here counteracts the effect of
  // cutlass::add_bias_and_interleave_int4s_inplace Input int8 data layout
  //      [elt_7  elt_5  elt_3  elt_1  elt_6  elt_4  elt_2  elt_0] (each elt
  //      occupies 4 bits)
  //
  // Converted fp16 data layout
  //      [elt_7  elt_6  elt_5  elt_4  elt_3  elt_2  elt_1  elt_0] (each elt
  //      occupies 16 bits)
  static constexpr int kConvertCount = 8;
  // using Converter
  //     =
  //     cutlass::FastInterleavedAndBiasedNumericArrayConverter<cutlass::half_t,
  //     cutlass::uint4b_t, kConvertCount>;

  // Each warp completes the internal reduce and writes the [Batch * NPerBlock *
  // Interleave] results to the corresponding address in shared memory
  template <int Num, int WarpSize>
  __device__ __forceinline__ static void sync(float* res,
                                              float (*sm)[Num * kInterleave]) {
#pragma unroll
    for (int i = 0; i < Num; ++i) {
      res[i] += __shfl_xor_sync(~0, res[i], 16);
      res[i] += __shfl_xor_sync(~0, res[i], 8);
      res[i] += __shfl_xor_sync(~0, res[i], 1);
    }
    __syncthreads();
    int warp = threadIdx.x / WarpSize, lane = threadIdx.x % WarpSize;
    if (lane == 0 || lane == 2 || lane == 4 || lane == 6) {
#pragma unroll
      for (int i = 0; i < Num; ++i) {
        sm[warp][i * kInterleave + lane / 2] = res[i];
      }
    }
    __syncthreads();
  }
};

template <>
struct WeightLayoutDetails<WeightOnlyQuantType::Int8b> {
  // Every two rows of the original weights are interleaved into a row with
  // stride of 64, so if each thread processes 16 elements(for int8, we can use
  // ldg.128 to load weights), then every group of four adjacent threads will
  // alternately process two different row weights for example every 128
  // consecutive int8 elements [128*i, 128*(i+1)-1] of row N under interleave
  // layout, the first 64 are from [64*i, 64*(i+1)-1] of row 2N before
  // interleaving, and the last 64 are from [64*i, 64*(i+1)-1] of row 2N+1
  // before interleaving. So if each thread loads 16 int8 elements, then the
  // elements of the first four and last four threads of each 8 consecutive
  // threads will come from row 2N and row 2N+1 respectively before
  // interleaving.
  static constexpr int kElemBits = 8;
  static constexpr int kInterleave = 2;
  static constexpr int kStride = 64;

  // The index remapping here is to counteracts the effect of
  // cutlass::permute_B_rows_for_mixed_gemm input 0 1 2 3 4 5 6 7 8 9 10 11 12
  // 13 14 15 weight 0 1 8 9 2 3 10 11 4 5 12 13 6 7 14 15
  static constexpr int kShuffleSize = 16;
  static constexpr int kShuffleBasicTile = 2;
  static constexpr int kShuffleContinous = 2;
  static constexpr int kShuffleStrided = 4;

  // The rearrangement here counteracts the effect of
  // cutlass::add_bias_and_interleave_int8s_inplace Input int8 data layout
  //      [elt_3  elt_1  elt_2  elt_0] (each elt occupies 8 bits)
  //
  // Converted fp16 data layout
  //      [elt_3  elt_2  elt_1  elt_0] (each elt occupies 16 bits)
  static constexpr int kConvertCount = 4;
  // using Converter =
  // cutlass::FastInterleavedAndBiasedNumericArrayConverter<cutlass::half_t,
  // uint8_t, kConvertCount>;

  // Each warp completes the internal reduce and writes the [Batch * NPerBlock *
  // Interleave] results to the corresponding address in shared memory
  template <int Num, int WarpSize>
  __device__ __forceinline__ static void sync(float* res,
                                              float (*sm)[Num * kInterleave]) {
#pragma unroll
    for (int i = 0; i < Num; ++i) {
      res[i] += __shfl_xor_sync(~0, res[i], 16);
      res[i] += __shfl_xor_sync(~0, res[i], 8);
      res[i] += __shfl_xor_sync(~0, res[i], 2);
      res[i] += __shfl_xor_sync(~0, res[i], 1);
    }
    __syncthreads();
    int warp = threadIdx.x / WarpSize, lane = threadIdx.x % WarpSize;
    if (lane == 0 || lane == 4) {
#pragma unroll
      for (int i = 0; i < Num; ++i) {
        sm[warp][i * kInterleave + lane / 4] = res[i];
      }
    }
    __syncthreads();
  }
};

template <WeightOnlyQuantType QType>
struct WeightOnlyKernelDetails {
  using Layout = WeightLayoutDetails<QType>;

  static constexpr int kElemBits = Layout::kElemBits;
  static constexpr int kInterleave = Layout::kInterleave;
  static constexpr int kStride = Layout::kStride;

  static constexpr int kShuffleSize = Layout::kShuffleSize;
  static constexpr int kShuffleBasicTile = Layout::kShuffleBasicTile;
  static constexpr int kShuffleContinous = Layout::kShuffleContinous;
  static constexpr int kShuffleStrided = Layout::kShuffleStrided;

  // using Converter = typename Layout::Converter;
  static constexpr int kConvertCount = Layout::kConvertCount;

  // Use ldg128 load data from global memory
  static constexpr int kAccessSize = 128;
  using AccessType = uint4;

  static constexpr int kElemsPerByte = 8 / kElemBits;
  static constexpr int kElemsPerThread = kAccessSize / kElemBits;
  static constexpr int kBytePerThread = kElemsPerThread / kElemsPerByte;
  static constexpr int kThreadsNumPerTile = kStride / kElemsPerThread;
  static constexpr int kThreadsNumPerInterleave =
      kThreadsNumPerTile * kInterleave;

  static constexpr int kConvertIters = kElemsPerThread / kConvertCount;

  // Each thread loads 16(int8b)/32(int4b) quantized weight elements each time
  // through ldg128 So more times of ldg128 are needed to load the same number
  // of fp16 activation elements.
  static constexpr int kActivationElemNumPerAccess =
      kAccessSize / (sizeof(half) * 8);
  static constexpr int kActivationAccessNum =
      kElemsPerThread / kActivationElemNumPerAccess;
};

struct WeightOnlyPerChannel;
template <int GS>
struct WeightOnlyGroupWise;

template <typename WeightOnlyFlag>
struct WeightOnlyProperties;

template <>
struct WeightOnlyProperties<WeightOnlyPerChannel> {
  static constexpr bool kIsFineGrained = false;
  static constexpr int kGroupSize = 0;
};

template <int GS>
struct WeightOnlyProperties<WeightOnlyGroupWise<GS>> {
  static constexpr bool kIsFineGrained = true;
  static constexpr int kGroupSize = GS;
};

template <typename T,
          WeightOnlyQuantType QType,
          typename WeightOnlyFlag,
          bool Zero,
          int BlockSize>
struct WeightOnlyScaleLoader {
  using Details = WeightOnlyKernelDetails<QType>;
  static constexpr bool kIsFineGrained =
      WeightOnlyProperties<WeightOnlyFlag>::kIsFineGrained;
  static constexpr int kGroupSize =
      WeightOnlyProperties<WeightOnlyFlag>::kGroupSize;

 private:
  const T* _scales;
  const T* _zeros;
  int _stride;
  int _offset;

 public:
  __device__ __forceinline__ WeightOnlyScaleLoader(const T* scales,
                                                   const T* zeros,
                                                   int initial_offset,
                                                   int stride)
      : _scales(scales), _zeros(zeros), _stride(stride) {
    _scales += initial_offset;
    if (Zero) {
      _zeros += initial_offset;
    }
    // Calculate the k dimension index of the element processed by the current
    // thread of layout before interleave Used to load scales and zeros in
    // groupwise weight only quant
    _offset =
        threadIdx.x / Details::kThreadsNumPerInterleave * Details::kStride +
        (threadIdx.x % Details::kThreadsNumPerTile) * Details::kElemsPerThread;
  }

  __device__ __forceinline__ void load(T* scale, T* zero, int nid) {
    int offset = nid * Details::kInterleave;

    if (kIsFineGrained) {
      offset += _offset / kGroupSize * _stride;
    }
    *scale = _scales[offset];

    if (Zero) {
      *zero = _zeros[offset];
    } else {
      *zero = static_cast<T>(0.f);
    }
  }

  __device__ __forceinline__ void advance() {
    _offset += BlockSize * Details::kElemsPerThread / Details::kInterleave;
  }

  __device__ __forceinline__ int offset() { return _offset; }
};

template <typename T, WeightOnlyQuantType QType>
struct WeightOnlyConverter {};

template <>
struct WeightOnlyConverter<half, WeightOnlyQuantType::Int8b> {
  static __device__ inline void convert(half halves[4],
                                        int8_t signed_chars[4]) {
    uint32_t* h = reinterpret_cast<uint32_t*>(halves);
    uint32_t i8s = *reinterpret_cast<uint32_t*>(signed_chars);

    static constexpr uint32_t mask_for_elt_01 = 0x5150;
    static constexpr uint32_t mask_for_elt_23 = 0x5352;
    static constexpr uint32_t start_byte_for_fp16 = 0x64646464;
    asm volatile("prmt.b32 %0,%1,%2,%3;\n"
                 : "=r"(h[0])
                 : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_01));
    asm volatile("prmt.b32 %0,%1,%2,%3;\n"
                 : "=r"(h[1])
                 : "r"(i8s), "n"(start_byte_for_fp16), "n"(mask_for_elt_23));

    static constexpr uint32_t I8s_TO_F16s_MAGIC_NUM = 0x64806480;
    asm volatile("sub.f16x2 %0, %1, %2;\n"
                 : "=r"(h[0])
                 : "r"(h[0]), "r"(I8s_TO_F16s_MAGIC_NUM));
    asm volatile("sub.f16x2 %0, %1, %2;\n"
                 : "=r"(h[1])
                 : "r"(h[1]), "r"(I8s_TO_F16s_MAGIC_NUM));
  }
};

template <>
struct WeightOnlyConverter<__hip_bfloat16, WeightOnlyQuantType::Int8b> {
  static __device__ inline void convert(__hip_bfloat16 halves[4],
                                        int8_t signed_chars[4]) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    uint32_t* bf16_result_ptr = reinterpret_cast<uint32_t*>(halves);
    uint32_t i8s = *reinterpret_cast<uint32_t*>(signed_chars);

    static constexpr uint32_t fp32_base = 0x4B000000;
    float fp32_intermediates[4];

    // Construct FP32s, bfloat does not have enough mantissa for IADD trick
    uint32_t* fp32_intermediates_casted =
        reinterpret_cast<uint32_t*>(fp32_intermediates);
    fp32_intermediates_casted[0] = __byte_perm(i8s, fp32_base, 0x7650);
    fp32_intermediates_casted[1] = __byte_perm(i8s, fp32_base, 0x7651);
    fp32_intermediates_casted[2] = __byte_perm(i8s, fp32_base, 0x7652);
    fp32_intermediates_casted[3] = __byte_perm(i8s, fp32_base, 0x7653);

    // Subtract out fp32_base + 128 to make the unsigned integer signed.
#pragma unroll
    for (int ii = 0; ii < 4; ++ii) {
      fp32_intermediates[ii] -= 8388736.f;
    }

// Truncate the fp32 representation and pack up as bfloat16s.
#pragma unroll
    for (int ii = 0; ii < 2; ++ii) {
      bf16_result_ptr[ii] = __byte_perm(fp32_intermediates_casted[2 * ii + 0],
                                        fp32_intermediates_casted[2 * ii + 1],
                                        0x7632);
    }
#else
    // Disable this on architectures older than Ampere since they lack hardware
    // for bf16 mma. If one wishes to use HMMA on older hardware, they should
    // Convert directly to FP16 using FP16 converters.
    assert(false);
#endif
  }
};

template <>
struct WeightOnlyConverter<half, WeightOnlyQuantType::Int4b> {
  static __device__ inline void convert(half halves[8],
                                        int8_t signed_chars[4]) {
    uint32_t* h = reinterpret_cast<uint32_t*>(halves);
    uint32_t i4s = *reinterpret_cast<uint32_t*>(signed_chars);

    // First, we extract the i4s and construct an intermediate fp16 number.
    static constexpr uint32_t immLut = (0xf0 & 0xcc) | 0xaa;
    static constexpr uint32_t BOTTOM_MASK = 0x000f000f;
    static constexpr uint32_t TOP_MASK = 0x00f000f0;
    static constexpr uint32_t I4s_TO_F16s_MAGIC_NUM = 0x64006400;

    // Note that the entire sequence only requires 1 shift instruction. This is
    // thanks to the register packing format and the fact that we force our
    // integers to be unsigned, and account for this in the fp16 subtractions.
    // In addition, I exploit the fact that sub and fma have the same throughput
    // in order to convert elt_23 and elt_67 to fp16 without having to shift
    // them to the bottom bits before hand.

    // Shift right by 8 to now consider elt_45 and elt_67. Issue first to hide
    // RAW dependency if we issue immediately before required.
    const uint32_t top_i4s = i4s >> 8;
    // Extract elt_01 - (i4s & 0x000f000f) | 0x64006400
    asm volatile(
        "lop3.b32 %0, %1, %2, %3, %4;\n"
        : "=r"(h[0])
        : "r"(i4s), "n"(BOTTOM_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
    // Extract elt_23 (i4s & 0x00f000f0) | 0x64006400
    asm volatile(
        "lop3.b32 %0, %1, %2, %3, %4;\n"
        : "=r"(h[1])
        : "r"(i4s), "n"(TOP_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));
    // Extract elt_45 (top_i4s & 0x000f000f) | 0x64006400
    asm volatile("lop3.b32 %0, %1, %2, %3, %4;\n"
                 : "=r"(h[2])
                 : "r"(top_i4s),
                   "n"(BOTTOM_MASK),
                   "n"(I4s_TO_F16s_MAGIC_NUM),
                   "n"(immLut));
    // Extract elt_67 (top_i4s & 0x00f000f0) | 0x64006400
    asm volatile(
        "lop3.b32 %0, %1, %2, %3, %4;\n"
        : "=r"(h[3])
        : "r"(top_i4s), "n"(TOP_MASK), "n"(I4s_TO_F16s_MAGIC_NUM), "n"(immLut));

    // I use inline PTX below because I am not sure if the compiler will emit
    // float2half instructions if I use the half2 ctor. In this case, I chose
    // performance reliability over code readability.

    // This is the half2 {1032, 1032} represented as an integer.
    static constexpr uint32_t FP16_TOP_MAGIC_NUM = 0x64086408;
    // This is the half2 {1 / 16, 1 / 16} represented as an integer.
    static constexpr uint32_t ONE_SIXTEENTH = 0x2c002c00;
    // This is the half2 {-72, -72} represented as an integer.
    static constexpr uint32_t NEG_72 = 0xd480d480;
    // Finally, we construct the output numbers.
    // Convert elt_01
    asm volatile("sub.f16x2 %0, %1, %2;\n"
                 : "=r"(h[0])
                 : "r"(h[0]), "r"(FP16_TOP_MAGIC_NUM));
    // Convert elt_23
    asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n"
                 : "=r"(h[1])
                 : "r"(h[1]), "r"(ONE_SIXTEENTH), "r"(NEG_72));
    // Convert elt_45
    asm volatile("sub.f16x2 %0, %1, %2;\n"
                 : "=r"(h[2])
                 : "r"(h[2]), "r"(FP16_TOP_MAGIC_NUM));
    // Convert elt_67
    asm volatile("fma.rn.f16x2 %0, %1, %2, %3;\n"
                 : "=r"(h[3])
                 : "r"(h[3]), "r"(ONE_SIXTEENTH), "r"(NEG_72));
  }
};

template <>
struct WeightOnlyConverter<__hip_bfloat16, WeightOnlyQuantType::Int4b> {
  static __device__ inline void convert(__hip_bfloat16 halves[8],
                                        int8_t signed_chars[4]) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    uint32_t* h = reinterpret_cast<uint32_t*>(halves);
    uint32_t const source_i4s = *reinterpret_cast<uint32_t*>(signed_chars);

    static constexpr uint32_t immLut = (0xf0 & 0xcc) | 0xaa;
    static constexpr uint32_t MASK = 0x000f000f;
    static constexpr uint32_t I4s_TO_BF16s_MAGIC_NUM = 0x43004300;

    // We don't have enough mantissa to remove as much shift overhead as FP16,
    // so we must loop. No shift needed for first item.
    uint32_t i4s = source_i4s;
    asm volatile(
        "lop3.b32 %0, %1, %2, %3, %4;\n"
        : "=r"(h[0])
        : "r"(i4s), "n"(MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
#pragma unroll
    for (int ii = 1; ii < 4; ++ii) {
      i4s >>= 4;
      // (i4s & 0x000f000f) | 0x43004300
      asm volatile(
          "lop3.b32 %0, %1, %2, %3, %4;\n"
          : "=r"(h[ii])
          : "r"(i4s), "n"(MASK), "n"(I4s_TO_BF16s_MAGIC_NUM), "n"(immLut));
    }

    // This is the BF16 {-136, -136} represented as an integer.
    static constexpr uint32_t BF16_BIAS = 0xC308C308;
    static constexpr uint32_t BF16_ONE = 0x3F803F80;

// Finally, we construct the output numbers.
#pragma unroll
    for (int ii = 0; ii < 4; ++ii) {
      // Since this section is for Ampere+, we use bf16 fma to do the bias
      // subtraction
      asm("fma.rn.bf16x2 %0, %1, %2, %3;\n"
          : "=r"(h[ii])
          : "r"(h[ii]), "r"(BF16_ONE), "r"(BF16_BIAS));
    }

#else
    // Disable this on architectures older than Ampere since they lack hardware
    // for bf16 mma. If one wishes to use HMMA on older hardware, they should
    // Convert directly to FP16 using FP16 converters.
    assert(false);
#endif
  }
};

template <typename VecType, typename T0, typename T1>
__device__ __forceinline__ void load(T0* dst, T1* src, size_t offset = 0) {
  *reinterpret_cast<VecType*>(dst) =
      *(reinterpret_cast<const VecType*>(src) + offset);
}

template <typename T, WeightOnlyQuantType QType, typename Details>
struct WeightPostProcessor {
  static __device__ __forceinline__ void run(T* weights_vec,
                                             T* weights_f16,
                                             T* scale,
                                             T* zero,
                                             int NPerBlock,
                                             int idx) {}
};

template <typename T, typename Details>
struct WeightPostProcessor<T, WeightOnlyQuantType::Int4b, Details> {
  static __device__ __forceinline__ void run(T* weights_vec,
                                             T* weights_f16,
                                             T* scale,
                                             T* zero,
                                             int NPerBlock,
                                             int idx) {
    using HALF_2_TYPE = typename CUDA_HALF_2_TYPE_TARIS<T>::type;
#pragma unroll
    for (int i = 0; i < Details::kShuffleContinous; ++i) {
#pragma unroll
      for (int j = 0; j < Details::kShuffleStrided; ++j) {
        // Dequantize the weights and arrange the shuffled elements back to
        // the correct order in the register array
        HALF_2_TYPE v = *reinterpret_cast<HALF_2_TYPE*>(
            weights_vec + i * Details::kShuffleBasicTile +
            j * Details::kShuffleContinous * Details::kShuffleBasicTile);
        v = __hfma2(v,
                    ConvertDstFunc_2<HALF_2_TYPE>::apply(scale[idx]),
                    ConvertDstFunc_2<HALF_2_TYPE>::apply(zero[idx]));
        weights_f16[(i * Details::kShuffleStrided * Details::kShuffleBasicTile +
                     j * Details::kShuffleBasicTile + 0) *
                        NPerBlock +
                    idx] = v.x;
        weights_f16[(i * Details::kShuffleStrided * Details::kShuffleBasicTile +
                     j * Details::kShuffleBasicTile + 1) *
                        NPerBlock +
                    idx] = v.y;
#ifdef _DEBUG_WEIGHT_ONLY_GEMV
        if (threadIdx.x == 0 && blockIdx.x == 0) {
          printf(
              "int4 weights_f16_idx: %d, weights_vec_idx: %d\n",
              (i * Details::kShuffleStrided * Details::kShuffleBasicTile +
               j * Details::kShuffleBasicTile + 0) *
                      NPerBlock +
                  idx,
              i * Details::kShuffleBasicTile +
                  j * Details::kShuffleContinous * Details::kShuffleBasicTile);
        }
#endif
      }
    }
  }
};

template <typename T, typename Details>
struct WeightPostProcessor<T, WeightOnlyQuantType::Int8b, Details> {
  static __device__ __forceinline__ void run(T* weights_vec,
                                             T* weights_f16,
                                             T* scale,
                                             T* zero,
                                             int NPerBlock,
                                             int idx) {
#pragma unroll
    for (int p = 0; p < 16; ++p) {
      weights_f16[p * NPerBlock + idx] =
          weights_vec[p / 8 + (p % 8) * 2] * scale[idx];
#ifdef _DEBUG_WEIGHT_ONLY_GEMV
      if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("int8 weights_f16_idx: %d, weights_vec_idx: %d\n",
               p * NPerBlock + idx,
               p / 8 + (p % 8) * 2);
      }
#endif
    }
  }
};

template <typename T,
          WeightOnlyQuantType QType,
          typename WeightOnlyFlag,
          bool Gelu,
          bool Zero,
          bool Bias,
          int NPerBlock,
          int Batch,
          int BlockSize>
__global__ void weight_only_batched_gemv_multi_warp(const T* in,
                                                    const int8_t* qweight,
                                                    const T* bias,
                                                    const T* scales,
                                                    const T* zeros,
                                                    T* out,
                                                    const int n,
                                                    const int k) {
  static_assert(NPerBlock == 1 || (NPerBlock % 2 == 0),
                "NPerBlock must be 1 or even in gemv multi warp kernel. ");
  using Details = WeightOnlyKernelDetails<QType>;

  // using Converter = typename Details::Converter;
  using AccType = typename Details::AccessType;
  using CvtSrcType = int8_t;
  using CvtResType = T;
  using ScaleLoader =
      WeightOnlyScaleLoader<T, QType, WeightOnlyFlag, Zero, BlockSize>;
  using WeightProcessor = WeightPostProcessor<T, QType, Details>;

  extern __shared__ uint8_t shmem[];
  constexpr int Interleave = Details::kInterleave;
  constexpr int WarpSize = 32;
  constexpr int Num = Batch * NPerBlock;
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int n_start_id = bid * NPerBlock * Interleave;
  using HALF_2_TYPE = typename CUDA_HALF_2_TYPE_TARIS<T>::type;
  // Calculate the n-dimensional index of the data processed by the current
  // thread in the interleave tile
  const int interleave_n_id = (tid / Details::kThreadsNumPerTile) % Interleave;

  qweight += n_start_id * k / Details::kElemsPerByte;
  ScaleLoader scale_loader(scales, zeros, n_start_id + interleave_n_id, n);

  float(*sm)[Num * Interleave] =
      reinterpret_cast<float(*)[Num * Interleave]>(shmem);

  // In order to take advantage of hfma2, we use fp16 for accumulation within
  // threads and fp32 for accumulation between threads.
  T accumulator[Num];
  for (int i = 0; i < Num; ++i) {
    accumulator[i] = ConvertDstFunc<T>::apply(0.f);
  }

  // Iteration in k dimensions
  for (int local_k = tid * Details::kElemsPerThread; local_k < k * Interleave;
       local_k += BlockSize * Details::kElemsPerThread) {
    T weights_f16[Details::kElemsPerThread * NPerBlock];
    T scale[NPerBlock], zero[NPerBlock];
#pragma unroll
    for (int idx = 0; idx < NPerBlock; ++idx) {
      // Load quantized weight and scales/zeros
      int8_t weights_quantized[Details::kBytePerThread];
      load<AccType>(weights_quantized,
                    qweight + idx * Interleave * k / Details::kElemsPerByte +
                        local_k / Details::kElemsPerByte);
      scale_loader.load(scale + idx, zero + idx, idx);
      T weights_vec[Details::kElemsPerThread];

#pragma unroll
      for (int i = 0; i < Details::kConvertIters; ++i) {
        // Use cutlass::FastInterleavedAndBiasedNumericArrayConverter for I2F
        // type conversion
        WeightOnlyConverter<T, QType>::convert(
            weights_vec + i * Details::kConvertCount,
            weights_quantized +
                i * Details::kConvertCount / Details::kElemsPerByte);
      }
      // Assign weight and apply scales.
      // Currently not support zero.
      WeightProcessor::run(
          weights_vec, weights_f16, scale, zero, NPerBlock, idx);
    }
#pragma unroll
    for (int b = 0; b < Batch; ++b) {
      T in_v[Details::kElemsPerThread];
#pragma unroll
      for (int idx = 0; idx < Details::kActivationAccessNum; ++idx) {
        load<AccType>(in_v + idx * Details::kActivationElemNumPerAccess,
                      in + b * k + scale_loader.offset() +
                          idx * Details::kActivationElemNumPerAccess);
      }
      // Perform vector inner product and accumulate
#ifndef WIN32
      if constexpr (NPerBlock == 1) {
#else
      if (NPerBlock == 1) {
#endif
        HALF_2_TYPE v = ConvertDstFunc<HALF_2_TYPE>::apply(0.f);
#pragma unroll
        for (int y = 0; y < Details::kElemsPerThread; y += 2) {
          v = HalfMulAdd<HALF_2_TYPE>::apply(
              *reinterpret_cast<HALF_2_TYPE*>(weights_f16 + y),
              *reinterpret_cast<HALF_2_TYPE*>(in_v + y),
              v);
        }
        accumulator[b] += ConvertDstFunc<T>::apply(v.x + v.y);
      } else {
#pragma unroll
        for (int x = 0; x < NPerBlock / 2; ++x) {
#pragma unroll
          for (int y = 0; y < Details::kElemsPerThread; ++y) {
            *reinterpret_cast<HALF_2_TYPE*>(accumulator + b * NPerBlock +
                                            x * 2) =
                HalfMulAdd<HALF_2_TYPE>::apply(
                    *reinterpret_cast<HALF_2_TYPE*>(weights_f16 +
                                                    y * NPerBlock + x * 2),
                    ConvertDstFunc_2<HALF_2_TYPE>::apply(in_v[y]),
                    *reinterpret_cast<HALF_2_TYPE*>(accumulator +
                                                    b * NPerBlock + x * 2));
          }
        }
      }
    }
    scale_loader.advance();
  }
  float reses[Num];
#pragma unroll
  for (int i = 0; i < Num; ++i) {
    reses[i] = ConvertFloatFunc<T>::apply(accumulator[i]);
  }

  // Each warp completes the internal reduce and writes the [Batch * NPerBlock *
  // Interleave] results to the corresponding address in shared memory
  Details::Layout::sync<Num, WarpSize>(reses, sm);

  // Each thread is responsible for the accumulation and store to global memory
  // of one element
  for (int i = tid; i < Num * Interleave; i += BlockSize) {
    int nid = i % (NPerBlock * Interleave);
    float v = 0.f;
    for (int j = 0; j < BlockSize / WarpSize; ++j) {
      v += sm[j][i];
    }
    float bias_v = 0.f;
#ifndef WIN32
    if constexpr (Bias) {
#else
    if (Bias) {
#endif
      bias_v = ConvertFloatFunc<T>::apply(bias[n_start_id + nid]);
    }
    int b = i / NPerBlock / Interleave;
    out[b * n + n_start_id + nid] = ConvertDstFunc<T>::apply(
        GeluActivation<float, Gelu>::apply(v + bias_v));
  }
}
#endif

template <typename T,
          WeightOnlyQuantType QType,
          typename WeightOnlyFlag,
          int NPerBlock,
          int Batch,
          int BlockSize>
void select_activation_and_bias(const T* input,
                                const int8_t* weight,
                                const T* bias,
                                const T* scales,
                                const int m,
                                const int n,
                                const int k,
                                const std::string& act_method,
                                T* output,
                                hipStream_t stream) {
#ifdef PADDLE_WITH_CUDA
  static constexpr int kInterleave = WeightLayoutDetails<QType>::kInterleave;
  dim3 grid(n / NPerBlock / kInterleave);
  dim3 block(BlockSize);
  int size = sizeof(float) * BlockSize / 32 * Batch * NPerBlock * kInterleave;
  if (bias) {
    if (act_method == "gelu") {
      weight_only_batched_gemv_multi_warp<T,
                                          QType,
                                          WeightOnlyFlag,
                                          true,
                                          false,
                                          true,
                                          NPerBlock,
                                          Batch,
                                          BlockSize>
          <<<grid, block, size, stream>>>(
              input, weight, bias, scales, /*zeros*/ nullptr, output, n, k);
    } else if (act_method == "None") {
      weight_only_batched_gemv_multi_warp<T,
                                          QType,
                                          WeightOnlyFlag,
                                          false,
                                          false,
                                          true,
                                          NPerBlock,
                                          Batch,
                                          BlockSize>
          <<<grid, block, size, stream>>>(
              input, weight, bias, scales, /*zeros*/ nullptr, output, n, k);
    } else {
      PADDLE_THROW(
          errors::InvalidArgument("Currently, weightonly GEMV act_method "
                                  "only support `gelu`, `None`. "));
    }
  } else {
    if (act_method == "gelu") {
      weight_only_batched_gemv_multi_warp<T,
                                          QType,
                                          WeightOnlyFlag,
                                          true,
                                          false,
                                          false,
                                          NPerBlock,
                                          Batch,
                                          BlockSize>
          <<<grid, block, size, stream>>>(
              input, weight, bias, scales, /*zeros*/ nullptr, output, n, k);
    } else if (act_method == "None") {
      weight_only_batched_gemv_multi_warp<T,
                                          QType,
                                          WeightOnlyFlag,
                                          false,
                                          false,
                                          false,
                                          NPerBlock,
                                          Batch,
                                          BlockSize>
          <<<grid, block, size, stream>>>(
              input, weight, bias, scales, /*zeros*/ nullptr, output, n, k);
    } else {
      PADDLE_THROW(
          errors::InvalidArgument("Currently, weightonly GEMV act_method "
                                  "only support `gelu`, `None`. "));
    }
  }
#endif
}

template <typename T, typename WeightOnlyFlag>
void weight_only_batched_gemv_launcher(
    const T* input,
    const int8_t* weight,
    const T* bias,
    const T* scales,
    int m,
    int n,
    int k,
    const std::string& weight_only_quant_type,
    const std::string& act_method,
    T* output,
    hipStream_t stream) {
#ifdef PADDLE_WITH_CUDA
  if (weight_only_quant_type == "int4") {
    switch (m) {
      case 1: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int4b,
                                   WeightOnlyFlag,
                                   1,
                                   1,
                                   192>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      case 2: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int4b,
                                   WeightOnlyFlag,
                                   2,
                                   2,
                                   128>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      case 3: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int4b,
                                   WeightOnlyFlag,
                                   2,
                                   3,
                                   256>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      case 4: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int4b,
                                   WeightOnlyFlag,
                                   4,
                                   4,
                                   256>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      default: {
        throw std::runtime_error(
            "Weight only cuda kernel only supported bs <= 4");
        break;
      }
    }
  } else if (weight_only_quant_type == "int8") {
    switch (m) {
      case 1: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int8b,
                                   WeightOnlyFlag,
                                   2,
                                   1,
                                   256>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      case 2: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int8b,
                                   WeightOnlyFlag,
                                   2,
                                   2,
                                   256>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      case 3: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int8b,
                                   WeightOnlyFlag,
                                   2,
                                   3,
                                   256>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      case 4: {
        select_activation_and_bias<T,
                                   WeightOnlyQuantType::Int8b,
                                   WeightOnlyFlag,
                                   2,
                                   4,
                                   256>(
            input, weight, bias, scales, m, n, k, act_method, output, stream);
        break;
      }
      default: {
        throw std::runtime_error(
            "Weight only cuda kernel only supported bs <= 4");
        break;
      }
    }
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "WeightOnlyGemvKernel quant_type only support 'int4' or 'int8'."));
  }
#endif
}

}  // namespace

template <typename T, typename Context>
void WeightOnlyGemvWrapper(const Context& dev_ctx,
                           const T* input,
                           const int8_t* weight,
                           const T* bias,
                           const T* scales,
                           int m,
                           int n,
                           int k,
                           int group_size,
                           const std::string& weight_only_quant_type,
                           const std::string& weight_only_type,
                           const std::string& act_method,
                           T* output) {
  using DataType = typename PDDataTypeTraits<T>::DataType;
  if (weight_only_type == "per_channel") {
    PADDLE_ENFORCE_EQ(group_size,
                      -1,
                      phi::errors::InvalidArgument(
                          "group size must be -1 in per-channel mode."));

    weight_only_batched_gemv_launcher<DataType, WeightOnlyPerChannel>(
        reinterpret_cast<const DataType*>(input),
        reinterpret_cast<const int8_t*>(weight),
        reinterpret_cast<const DataType*>(bias),
        reinterpret_cast<const DataType*>(scales),
        m,
        n,
        k,
        weight_only_quant_type,
        act_method,
        reinterpret_cast<DataType*>(output),
        dev_ctx.stream());
  } else if (weight_only_type == "group_wise") {
    if (group_size == 64) {
      weight_only_batched_gemv_launcher<DataType, WeightOnlyGroupWise<64>>(
          reinterpret_cast<const DataType*>(input),
          reinterpret_cast<const int8_t*>(weight),
          reinterpret_cast<const DataType*>(bias),
          reinterpret_cast<const DataType*>(scales),
          m,
          n,
          k,
          weight_only_quant_type,
          act_method,
          reinterpret_cast<DataType*>(output),
          dev_ctx.stream());
    } else if (group_size == 128) {
      weight_only_batched_gemv_launcher<DataType, WeightOnlyGroupWise<128>>(
          reinterpret_cast<const DataType*>(input),
          reinterpret_cast<const int8_t*>(weight),
          reinterpret_cast<const DataType*>(bias),
          reinterpret_cast<const DataType*>(scales),
          m,
          n,
          k,
          weight_only_quant_type,
          act_method,
          reinterpret_cast<DataType*>(output),
          dev_ctx.stream());
    } else {
      PADDLE_THROW(phi::errors::InvalidArgument(
          "WeightOnlyGemvKernel group_size only support 64 or 128."));
    }
  } else {
    PADDLE_THROW(
        phi::errors::InvalidArgument("WeightOnlyGemvKernel type only support "
                                     "'per_channel' or 'group_wise'."));
  }
}

template <>
void WeightOnlyGemvWrapper(const phi::GPUContext& dev_ctx,
                           const float* input,
                           const int8_t* weight,
                           const float* bias,
                           const float* scales,
                           int m,
                           int n,
                           int k,
                           int group_size,
                           const std::string& weight_only_quant_type,
                           const std::string& weight_only_type,
                           const std::string& act_method,
                           float* output) {
  PADDLE_THROW(phi::errors::Unimplemented(
      "WeightOnlyGemvKernel type only support 'float16' and 'bfloa16."
      "Not support float32."));
}

template <typename T, typename Context>
void WeightOnlyGemvKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& weight,
                          const paddle::optional<DenseTensor>& bias,
                          const DenseTensor& weight_scale,
                          int group_size,
                          const std::string& weight_only_quant_type,
                          const std::string& weight_only_type,
                          const std::string& act_method,
                          DenseTensor* out) {
  const T* x_data = x.data<T>();
  const int8_t* weight_data = weight.data<int8_t>();
  // Actually, we pass the weight datatype is uint8_t type.
  const T* bias_data = bias ? bias.get().data<T>() : nullptr;
  const T* weight_scale_data = weight_scale.data<T>();
  T* out_data = dev_ctx.template Alloc<T>(out);
  int m = x.dims()[0];
  int k = x.dims()[1];
  int n = weight.dims()[0];

  WeightOnlyGemvWrapper<T>(dev_ctx,
                           x_data,
                           weight_data,
                           bias_data,
                           weight_scale_data,
                           m,
                           n,
                           k,
                           group_size,
                           weight_only_quant_type,
                           weight_only_type,
                           act_method,
                           out_data);
}

template void WeightOnlyGemvWrapper(const phi::GPUContext& ctx,
                                    const float* input,
                                    const int8_t* weight,
                                    const float* bias,
                                    const float* scales,
                                    int m,
                                    int n,
                                    int k,
                                    int group_size,
                                    const std::string& weight_only_quant_type,
                                    const std::string& weight_only_type,
                                    const std::string& act_method,
                                    float* output);

template void WeightOnlyGemvWrapper(const phi::GPUContext& ctx,
                                    const phi::dtype::float16* input,
                                    const int8_t* weight,
                                    const phi::dtype::float16* bias,
                                    const phi::dtype::float16* scales,
                                    int m,
                                    int n,
                                    int k,
                                    int group_size,
                                    const std::string& weight_only_quant_type,
                                    const std::string& weight_only_type,
                                    const std::string& act_method,
                                    phi::dtype::float16* output);

template void WeightOnlyGemvWrapper(const phi::GPUContext& ctx,
                                    const phi::dtype::bfloat16* input,
                                    const int8_t* weight,
                                    const phi::dtype::bfloat16* bias,
                                    const phi::dtype::bfloat16* scales,
                                    int m,
                                    int n,
                                    int k,
                                    int group_size,
                                    const std::string& weight_only_quant_type,
                                    const std::string& weight_only_type,
                                    const std::string& act_method,
                                    phi::dtype::bfloat16* output);
}  // namespace phi
