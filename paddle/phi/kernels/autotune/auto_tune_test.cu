#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>

#include "glog/logging.h"
#include "paddle/phi/api/lib/utils/allocator.h"
#include "paddle/phi/backends/all_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/autotune/auto_tune_base.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"

namespace tune = phi::autotune;

template <typename T, int VecSize>
__global__ void VecSumTest(const T* x, T* y, int N) {
#ifdef __HIPCC__
  int idx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
#else
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
#endif
  using LoadT = phi::AlignedVector<T, VecSize>;
  for (int i = idx * VecSize; i < N; i += blockDim.x * gridDim.x * VecSize) {
    LoadT x_vec;
    LoadT y_vec;
    phi::Load<T, VecSize>(&x[i], &x_vec);
    phi::Load<T, VecSize>(&y[i], &y_vec);
#pragma unroll
    for (int j = 0; j < VecSize; j++) {
      y_vec[j] = x_vec[j] + y_vec[j];
    }
    phi::Store<T, VecSize>(y_vec, &y[i]);
  }
}

template <int Vecsize>
float Algo(const phi::GPUContext& ctx,
           const phi::DenseTensor& d_in,
           phi::DenseTensor* d_out,
           size_t N,
           size_t threads,
           size_t blocks) {
  const float* d_in_data = d_in.data<float>();
  float* d_out_data = d_out->data<float>();
#ifdef __HIPCC__
  hipLaunchKernelGGL(HIP_KERNEL_NAME(VecSumTest<float, Vecsize>),
                     dim3(blocks),
                     dim3(threads),
                     0,
                     0,
                     d_in_data,
                     d_out_data,
                     N);
#else
  VLOG(3) << "Vecsize is " << Vecsize;
  VecSumTest<float, Vecsize>
      <<<blocks, threads, 0, ctx.stream()>>>(d_in_data, d_out_data, N);
#endif
  return Vecsize;
}

TEST(AutoTune, sum) {
  int64_t N = 1 << 20;
  size_t blocks = 512;
  size_t threads = 256;
  size_t size = sizeof(float) * N;

  const auto alloc_cpu =
      std::make_unique<paddle::experimental::DefaultAllocator>(
          paddle::platform::CPUPlace());
  auto in1 = std::make_shared<phi::DenseTensor>(
      alloc_cpu.get(),
      phi::DenseTensorMeta(
          phi::DataType::FLOAT32, phi::make_ddim({N}), phi::DataLayout::NCHW));
  auto in2 = std::make_shared<phi::DenseTensor>(
      alloc_cpu.get(),
      phi::DenseTensorMeta(
          phi::DataType::FLOAT32, phi::make_ddim({N}), phi::DataLayout::NCHW));

  float* in1_data = in1->data<float>();
  float* in2_data = in2->data<float>();
  for (size_t i = 0; i < N; i++) {
    in1_data[i] = 1.0f;
    in2_data[i] = 2.0f;
  }
#if defined(PADDLE_WITH_CUDA) || defined(PADDLE_WITH_HIP)
  const auto alloc_cuda =
      std::make_unique<paddle::experimental::DefaultAllocator>(
          paddle::platform::CUDAPlace());
  phi::DeviceContextPool& pool = phi::DeviceContextPool::Instance();
  auto place = paddle::platform::CUDAPlace();
  auto* dev_ctx = static_cast<const phi::GPUContext*>(pool.GetByPlace(place));
  auto stream = dev_ctx->stream();

  auto d_in1 = std::make_shared<phi::DenseTensor>(
      alloc_cuda.get(),
      phi::DenseTensorMeta(
          phi::DataType::FLOAT32, phi::make_ddim({N}), phi::DataLayout::NCHW));
  auto d_in2 = std::make_shared<phi::DenseTensor>(
      alloc_cuda.get(),
      phi::DenseTensorMeta(
          phi::DataType::FLOAT32, phi::make_ddim({N}), phi::DataLayout::NCHW));
  phi::Copy(*dev_ctx, *in1.get(), phi::GPUPlace(), false, d_in1.get());
  phi::Copy(*dev_ctx, *in2.get(), phi::GPUPlace(), false, d_in2.get());

  // 1. Test call_back.
  VLOG(3) << ">>> [CallBack]: Test case.";
  auto callback1 = tune::MakeCallback<float>(Algo<4>);
  auto callback2 = tune::MakeCallback<float>(Algo<2>);
  auto callback3 = tune::MakeCallback<float>(Algo<1>);
  std::vector<decltype(callback1)> callbacks{callback1, callback2, callback3};
  for (int i = 0; i < callbacks.size(); ++i) {
    dev_ctx->Wait();
    phi::GpuTimer timer;
    timer.Start(0);
    callbacks[i].Run(*dev_ctx, *d_in1.get(), d_in2.get(), N, threads, blocks);
    timer.Stop(0);
    VLOG(3) << "kernel[" << i << "]: time cost is " << timer.ElapsedTime();
  }
#endif
}
