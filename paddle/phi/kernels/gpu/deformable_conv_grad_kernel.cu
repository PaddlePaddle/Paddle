#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/deformable_conv_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/deformable_conv_grad_kernel_impl.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__global__ void ModulatedDeformableCol2imGpuKernel(
    const int nthreads,
    const T* data_col,
    const T* data_offset,
    const T* data_mask,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int deformable_group,
    const int height_col,
    const int width_col,
    T* grad_im) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t thread = index; thread < nthreads; thread += offset) {
    const int j = (thread / width_col / height_col / batch_size) % kernel_w;
    const int i =
        (thread / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c =
        thread / width_col / height_col / batch_size / kernel_w / kernel_h;

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = thread % width_col;
    int h_out = (thread / width_col) % height_col;
    int b = (thread / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const T* data_offset_ptr =
        data_offset + (b * deformable_group + deformable_group_index) * 2 *
                          kernel_h * kernel_w * height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const int data_mask_hw_ptr =
        ((i * kernel_w + j) * height_col + h_out) * width_col + w_out;
    const T offset_h = data_offset_ptr[data_offset_h_ptr];
    const T offset_w = data_offset_ptr[data_offset_w_ptr];
    const T cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const T cur_inv_w_data = w_in + j * dilation_w + offset_w;

    T cur_top_grad = data_col[thread];
    if (data_mask) {
      const T* data_mask_ptr =
          data_mask + (b * deformable_group + deformable_group_index) *
                          kernel_h * kernel_w * height_col * width_col;
      const T mask = data_mask_ptr[data_mask_hw_ptr];
      cur_top_grad *= mask;
    }
    const int cur_h = static_cast<int>(cur_inv_h_data);
    const int cur_w = static_cast<int>(cur_inv_w_data);
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          T weight = DmcnGetGradientWeight(cur_inv_h_data,
                                           cur_inv_w_data,
                                           cur_h + dy,
                                           cur_w + dx,
                                           height,
                                           width);

          paddle::platform::CudaAtomicAdd(grad_im + cur_bottom_grad_pos,
                                          weight * cur_top_grad);
        }
      }
    }
  }
}

template <typename T, typename Context>
void ModulatedDeformableCol2im(const Context& dev_ctx,
                               const T* data_col,
                               const T* data_offset,
                               const T* data_mask,
                               const std::vector<int64_t>& im_shape,
                               const std::vector<int64_t>& col_shape,
                               const std::vector<int64_t>& kernel_shape,
                               const std::vector<int>& pad,
                               const std::vector<int>& stride,
                               const std::vector<int>& dilation,
                               const int deformable_group,
                               T* grad_im) {
  int channel_per_deformable_group = im_shape[0] / deformable_group;
  int num_kernels = col_shape[0] * col_shape[1] * col_shape[2] * col_shape[3];
  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  ModulatedDeformableCol2imGpuKernel<T>
      <<<blocks, threads, 0, dev_ctx.stream()>>>(num_kernels,
                                                 data_col,
                                                 data_offset,
                                                 data_mask,
                                                 im_shape[0],
                                                 im_shape[1],
                                                 im_shape[2],
                                                 kernel_shape[2],
                                                 kernel_shape[3],
                                                 pad[0],
                                                 pad[1],
                                                 stride[0],
                                                 stride[1],
                                                 dilation[0],
                                                 dilation[1],
                                                 channel_per_deformable_group,
                                                 col_shape[1],
                                                 deformable_group,
                                                 col_shape[2],
                                                 col_shape[3],
                                                 grad_im);
}

template <typename T>
__global__ void ModulatedDeformableCol2imCoordGpuKernel(
    const int nthreads,
    const T* data_col,
    const T* data_im,
    const T* data_offset,
    const T* data_mask,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int offset_channels,
    const int deformable_group,
    const int height_col,
    const int width_col,
    T* grad_offset,
    T* grad_mask) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    T val = 0, mval = 0;
    const int w = i % width_col;
    const int h = (i / width_col) % height_col;
    const int c = (i / width_col / height_col) % offset_channels;
    const int b = (i / width_col / height_col) / offset_channels;

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const T* data_col_ptr = data_col + deformable_group_index *
                                           channel_per_deformable_group *
                                           batch_size * width_col * height_col;
    const T* data_im_ptr =
        data_im + (b * deformable_group + deformable_group_index) *
                      channel_per_deformable_group / kernel_h / kernel_w *
                      height * width;
    const T* data_offset_ptr =
        data_offset + (b * deformable_group + deformable_group_index) * 2 *
                          kernel_h * kernel_w * height_col * width_col;
    const T* data_mask_ptr =
        data_mask
            ? data_mask + (b * deformable_group + deformable_group_index) *
                              kernel_h * kernel_w * height_col * width_col
            : nullptr;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = offset_c / 2; col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos =
          (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i =
          (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const T offset_h = data_offset_ptr[data_offset_h_ptr];
      const T offset_w = data_offset_ptr[data_offset_w_ptr];
      T inv_h = h_in + i * dilation_h + offset_h;
      T inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -2;
      } else {
        mval += data_col_ptr[col_pos] *
                funcs::DmcnIm2colBilinear(data_im_ptr + cnt * height * width,
                                          width,
                                          height,
                                          width,
                                          inv_h,
                                          inv_w);
      }
      const T weight =
          DmcnGetCoordinateWeight(inv_h,
                                  inv_w,
                                  height,
                                  width,
                                  data_im_ptr + cnt * height * width,
                                  width,
                                  bp_dir);
      if (data_mask_ptr) {
        const int data_mask_hw_ptr =
            (((i * kernel_w + j) * height_col + h_out) * width_col + w_out);
        const T mask = data_mask_ptr[data_mask_hw_ptr];
        val += weight * data_col_ptr[col_pos] * mask;
      } else {
        val += weight * data_col_ptr[col_pos];
      }
      cnt += 1;
    }
    grad_offset[i] = val;
    if (grad_mask && offset_c % 2 == 0)
      grad_mask[(((b * deformable_group + deformable_group_index) * kernel_h *
                      kernel_w +
                  offset_c / 2) *
                     height_col +
                 h) *
                    width_col +
                w] = mval;
  }
}

template <typename T, typename Context>
void ModulatedDeformableCol2imCoord(const Context& dev_ctx,
                                    const T* data_col,
                                    const T* data_im,
                                    const T* data_offset,
                                    const T* data_mask,
                                    const std::vector<int64_t>& im_shape,
                                    const std::vector<int64_t>& col_shape,
                                    const std::vector<int64_t>& kernel_shape,
                                    const std::vector<int>& paddings,
                                    const std::vector<int>& strides,
                                    const std::vector<int>& dilations,
                                    const int deformable_groups,
                                    T* grad_offset,
                                    T* grad_mask) {
  int num_kernels = 2 * kernel_shape[2] * kernel_shape[3] * col_shape[1] *
                    col_shape[2] * col_shape[3] * deformable_groups;
  int channel_per_deformable_group = col_shape[0] / deformable_groups;
  int blocks = NumBlocks(num_kernels);
  int threads = kNumCUDAThreads;

  ModulatedDeformableCol2imCoordGpuKernel<T>
      <<<blocks, threads, 0, dev_ctx.stream()>>>(
          num_kernels,
          data_col,
          data_im,
          data_offset,
          data_mask,
          im_shape[0],
          im_shape[1],
          im_shape[2],
          kernel_shape[2],
          kernel_shape[3],
          paddings[0],
          paddings[1],
          strides[0],
          strides[1],
          dilations[0],
          dilations[1],
          channel_per_deformable_group,
          col_shape[1],
          2 * kernel_shape[2] * kernel_shape[3] * deformable_groups,
          deformable_groups,
          col_shape[2],
          col_shape[3],
          grad_offset,
          grad_mask);
}

template <typename T>
__global__ void FilterGradAddupGpuKernel(const int nthreads,
                                         const int n,
                                         const int height,
                                         const int width,
                                         const T* dweight_3d,
                                         T* filter_grad) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    filter_grad[i] = filter_grad[i] + dweight_3d[i];
  }
}

template <typename T, typename Context>
void FilterGradAddup(const Context& dev_ctx,
                     const int nthreads,
                     const int n,
                     const int height,
                     const int width,
                     const T* dweight_3d,
                     T* filter_grad) {
  FilterGradAddupGpuKernel<T>
      <<<NumBlocks(nthreads), kNumCUDAThreads, 0, dev_ctx.stream()>>>(
          nthreads, n, height, width, dweight_3d, filter_grad);
}

}  // namespace phi

PD_REGISTER_KERNEL(deformable_conv_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::DeformableConvGradKernel,
                   float,
                   double) {}
