#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/allclose_kernel.h"

#include "glog/logging.h"

#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T>
__global__ void AllcloseCUDAKernel(const T* in_data,
                                   const T* other_data,
                                   const double rtol,
                                   const double atol,
                                   bool equal_nan,
                                   int num,
                                   bool* out_data) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
  bool val;
  for (int i = idx; i < num; i += blockDim.x * gridDim.x) {
    const T a = in_data[i], b = other_data[i];
    if (isnan(a) || isnan(b)) {
      val = equal_nan && isnan(a) == isnan(b);
    } else {
      T left = (a > b ? a - b : b - a);
      T right = atol + (b > 0 ? rtol * b : (-rtol) * b);
      T diff = (left > right ? left - right : right - left);
      val = a == b || left <= right || diff <= 1e-15;
    }
    if (!val) *out_data = false;
  }
}

template <typename T, typename Context>
void AllCloseKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    const Scalar& rtol,
                    const Scalar& atol,
                    bool equal_nan,
                    DenseTensor* out) {
  double rtol_v, atol_v;
  if (rtol.dtype() == DataType::FLOAT64) {
    rtol_v = rtol.to<double>();
  } else if (rtol.dtype() == DataType::FLOAT32) {
    rtol_v = rtol.to<float>();
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Input (Rtol) type must be double or float, but get %s.",
        rtol.dtype()));
  }
  if (atol.dtype() == DataType::FLOAT64) {
    atol_v = atol.to<double>();
  } else if (atol.dtype() == DataType::FLOAT32) {
    atol_v = atol.to<float>();
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Input (Atol) type must be double or float, but get %s.",
        atol.dtype()));
  }
  VLOG(3) << "rtol and atol is : " << rtol_v << " " << atol_v;
  const T* in_data = x.data<T>();
  const T* other_data = y.data<T>();
  bool* out_data = dev_ctx.template Alloc<bool>(out);

  int num = x.numel();
  int block = 1024;
  int grid = (block - 1 + num) / block;
  grid = (grid > block) ? block : grid;
#ifdef PADDLE_WITH_HIP
  hipMemset(out_data, true, sizeof(bool));
#else
  hipMemset(out_data, true, sizeof(bool));
#endif
  AllcloseCUDAKernel<T><<<grid, block, 0, dev_ctx.stream()>>>(
      in_data, other_data, rtol_v, atol_v, equal_nan, num, out_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    allclose, GPU, ALL_LAYOUT, phi::AllCloseKernel, float, double) {
  kernel->OutputAt(0).SetDataType(phi::DataType::BOOL);
}
