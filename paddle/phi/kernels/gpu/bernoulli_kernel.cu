#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/random.h>
#include <thrust/transform.h>
#ifdef __NVCC__
#include <hiprand/hiprand_kernel.h>
#endif
#ifdef __HIPCC__
#include <hiprand_kernel.h>
#endif

#include <algorithm>
#include <vector>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/bernoulli_kernel.h"
#include "paddle/phi/kernels/funcs/distribution_helper.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/platform/transform.h"

DECLARE_bool(use_curand);

namespace phi {

template <typename T>
struct BernoulliCudaFunctor {
  unsigned int seed_;
  unsigned int offset_;
  __host__ __device__ BernoulliCudaFunctor(unsigned int seed,
                                           unsigned int offset)
      : seed_(seed), offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n, const T p) const {
    // NOTE(zhiqiu): currently, PADDLE_ENFORCE in cuda kernel may print several
    // lines of error messages if, and it should be refined.
    PADDLE_ENFORCE(p >= 0.0 && p <= 1.0,
                   "The probability should be >=0 and <= 1, but got %f",
                   p);
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(0.0, 1.0);
    rng.discard(n + offset_);
    return static_cast<T>(dist(rng) < p);
  }
};

// 'hiprand_uniform4/hiprand_uniform4' generate 4 random number each time
template <typename T>
__global__ void bernoulli_cuda_kernel(
    size_t size, uint64_t seed, uint64_t offset, const T* x_data, T* out_data) {
  size_t thread_idx =
      static_cast<size_t>(blockIdx.x * blockDim.x + threadIdx.x);

#if defined(__NVCC__)
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, thread_idx, offset, &state);
#else
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, thread_idx, offset, &state);
#endif

  size_t total_thread = gridDim.x * blockDim.x;
  for (size_t i = 4 * thread_idx; i < size; i += total_thread * 4) {
    funcs::uniform_distribution<float> dist;
    float4 rand = dist(&state);
#pragma unroll
    for (size_t j = 0; j < 4; j++) {
      size_t idx = i + j;
      if (idx < size) {
        out_data[idx] = static_cast<T>((&rand.x)[j] <= x_data[idx]);
      }
    }
  }
}

template <typename T, typename Context>
void BernoulliKernel(const Context& ctx,
                     const DenseTensor& x,
                     DenseTensor* out) {
  const T* x_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(out);
  auto numel = x.numel();

  auto gen_cuda = ctx.GetGenerator();

  if (FLAGS_use_curand) {
    auto seed_offset = gen_cuda->IncrementOffset(12);
    uint64_t seed = seed_offset.first;
    uint64_t offset = seed_offset.second;

    auto gpu_config = phi::backends::gpu::GetGpuLaunchConfig1D(ctx, numel, 4);
    size_t grid_size = gpu_config.GetGridSize();
    size_t block_size = gpu_config.GetBlockSize();

    bernoulli_cuda_kernel<<<grid_size, block_size, 0, ctx.stream()>>>(
        numel, seed, offset, x_data, out_data);
  } else {
    auto seed_offset = gen_cuda->IncrementOffset(1);
    int64_t gen_offset = numel * seed_offset.second;
    paddle::platform::Transform<phi::GPUContext> trans;
    thrust::counting_iterator<int64_t> index_sequence_begin(0);
    trans(ctx,
          index_sequence_begin,
          index_sequence_begin + numel,
          x_data,
          out_data,
          BernoulliCudaFunctor<T>(static_cast<int64_t>(seed_offset.first),
                                  static_cast<int64_t>(gen_offset)));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(
    bernoulli, GPU, ALL_LAYOUT, phi::BernoulliKernel, float, double) {}
