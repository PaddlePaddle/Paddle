#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/graph_reindex_kernel.h"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/graph_reindex_funcs.h"

namespace phi {

constexpr int WARP_SIZE = 32;

template <typename T, typename Context>
void FillHashTable(const Context& dev_ctx,
                   const T* input,
                   int num_input,
                   int64_t len_hashtable,
                   thrust::device_vector<T>* unique_items,
                   T* keys,
                   int* values,
                   int* key_index) {
#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int grid_tmp = (num_input + block - 1) / block;
  int grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  // Insert data into keys and values.
  BuildHashTable<T><<<grid, block, 0, dev_ctx.stream()>>>(
      input, num_input, len_hashtable, keys, key_index);

  // Get item index count.
  thrust::device_vector<int> item_count(num_input + 1, 0);
  GetItemIndexCount<T><<<grid, block, 0, dev_ctx.stream()>>>(
      input,
      thrust::raw_pointer_cast(item_count.data()),
      num_input,
      len_hashtable,
      keys,
      key_index);

  thrust::exclusive_scan(
      item_count.begin(), item_count.end(), item_count.begin());
  size_t total_unique_items = item_count[num_input];
  unique_items->resize(total_unique_items);

  // Get unique items
  FillUniqueItems<T><<<grid, block, 0, dev_ctx.stream()>>>(
      input,
      num_input,
      len_hashtable,
      thrust::raw_pointer_cast(unique_items->data()),
      thrust::raw_pointer_cast(item_count.data()),
      keys,
      values,
      key_index);
}

template <typename T, typename Context>
void FillBufferHashTable(const Context& dev_ctx,
                         const T* input,
                         int num_input,
                         thrust::device_vector<T>* unique_items,
                         int* values,
                         int* key_index) {
#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int grid_tmp = (num_input + block - 1) / block;
  int grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  // Insert data.
  BuildHashTable<T>
      <<<grid, block, 0, dev_ctx.stream()>>>(input, num_input, key_index);

  // Get item index count.
  thrust::device_vector<int> item_count(num_input + 1, 0);
  GetItemIndexCount<T><<<grid, block, 0, dev_ctx.stream()>>>(
      input, thrust::raw_pointer_cast(item_count.data()), num_input, key_index);

  thrust::exclusive_scan(
      item_count.begin(), item_count.end(), item_count.begin());
  size_t total_unique_items = item_count[num_input];
  unique_items->resize(total_unique_items);

  // Get unique items
  FillUniqueItems<T><<<grid, block, 0, dev_ctx.stream()>>>(
      input,
      num_input,
      thrust::raw_pointer_cast(unique_items->data()),
      thrust::raw_pointer_cast(item_count.data()),
      values,
      key_index);
}

template <typename T, typename Context>
void ResetBufferHashTable(const Context& dev_ctx,
                          const T* input,
                          int num_input,
                          thrust::device_vector<T>* unique_items,
                          int* values,
                          int* key_index) {
#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int grid_tmp = (unique_items->size() + block - 1) / block;
  int grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  ResetHashTable<T><<<grid, block, 0, dev_ctx.stream()>>>(
      thrust::raw_pointer_cast(unique_items->data()),
      unique_items->size(),
      key_index,
      values);
}

template <typename T, typename Context>
void Reindex(const Context& dev_ctx,
             const T* inputs,
             thrust::device_ptr<T> src_outputs,
             thrust::device_vector<T>* out_nodes,
             int num_inputs,
             int num_edges) {
  out_nodes->resize(num_inputs + num_edges);
  thrust::copy(inputs, inputs + num_inputs, out_nodes->begin());
  thrust::copy(
      src_outputs, src_outputs + num_edges, out_nodes->begin() + num_inputs);
  thrust::device_vector<T> unique_nodes;
  unique_nodes.clear();

  // Fill hash table
  int64_t num = out_nodes->size();
  int64_t log_num = 1 << static_cast<size_t>(1 + std::log2(num >> 1));
  int64_t table_size = log_num << 1;
  T* keys;
  int *values, *key_index;

#ifdef PADDLE_WITH_HIP
  hipMalloc(&keys, table_size * sizeof(T));
  hipMalloc(&values, table_size * sizeof(int));
  hipMalloc(&key_index, table_size * sizeof(int));
  hipMemset(keys, -1, table_size * sizeof(T));
  hipMemset(values, -1, table_size * sizeof(int));
  hipMemset(key_index, -1, table_size * sizeof(int));
#else
  hipMalloc(&keys, table_size * sizeof(T));
  hipMalloc(&values, table_size * sizeof(int));
  hipMalloc(&key_index, table_size * sizeof(int));
  hipMemset(keys, -1, table_size * sizeof(T));
  hipMemset(values, -1, table_size * sizeof(int));
  hipMemset(key_index, -1, table_size * sizeof(int));
#endif

  FillHashTable<T, Context>(dev_ctx,
                            thrust::raw_pointer_cast(out_nodes->data()),
                            out_nodes->size(),
                            table_size,
                            &unique_nodes,
                            keys,
                            values,
                            key_index);
  out_nodes->resize(unique_nodes.size());
  thrust::copy(unique_nodes.begin(), unique_nodes.end(), out_nodes->begin());

// Fill outputs with reindex result.
#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int grid_tmp = (num_edges + block - 1) / block;
  int grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  ReindexSrcOutput<T><<<grid, block, 0, dev_ctx.stream()>>>(
      thrust::raw_pointer_cast(src_outputs),
      num_edges,
      table_size,
      keys,
      values);
#ifdef PADDLE_WITH_HIP
  hipFree(keys);
  hipFree(values);
  hipFree(key_index);
#else
  hipFree(keys);
  hipFree(values);
  hipFree(key_index);
#endif
}

template <typename T, typename Context>
void BufferReindex(const Context& dev_ctx,
                   const T* inputs,
                   thrust::device_ptr<T> src_outputs,
                   thrust::device_vector<T>* out_nodes,
                   int num_inputs,
                   int* hashtable_value,
                   int* hashtable_index,
                   int num_edges) {
  out_nodes->resize(num_inputs + num_edges);
  thrust::copy(inputs, inputs + num_inputs, out_nodes->begin());
  thrust::copy(
      src_outputs, src_outputs + num_edges, out_nodes->begin() + num_inputs);
  thrust::device_vector<T> unique_nodes;
  unique_nodes.clear();

  // Fill hash table
  FillBufferHashTable<T, Context>(dev_ctx,
                                  thrust::raw_pointer_cast(out_nodes->data()),
                                  out_nodes->size(),
                                  &unique_nodes,
                                  hashtable_value,
                                  hashtable_index);
  out_nodes->resize(unique_nodes.size());
  thrust::copy(unique_nodes.begin(), unique_nodes.end(), out_nodes->begin());

// Fill outputs with reindex result.
#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int max_grid_dimx = dev_ctx.GetCUDAMaxGridDimSize()[0];
  int grid_tmp = (num_edges + block - 1) / block;
  int grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  ReindexSrcOutput<T><<<grid, block, 0, dev_ctx.stream()>>>(
      thrust::raw_pointer_cast(src_outputs), num_edges, hashtable_value);

  ResetBufferHashTable<T, Context>(dev_ctx,
                                   thrust::raw_pointer_cast(out_nodes->data()),
                                   out_nodes->size(),
                                   &unique_nodes,
                                   hashtable_value,
                                   hashtable_index);
}

template <typename T, int BLOCK_WARPS, int TILE_SIZE>
__global__ void GetDstEdgeCUDAKernel(const int64_t num_rows,
                                     const int* in_rows,
                                     const int* dst_counts,
                                     const int* dst_ptr,
                                     T* dst_outputs) {
  assert(blockDim.x == WARP_SIZE);
  assert(blockDim.y == BLOCK_WARPS);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  while (out_row < last_row) {
    const int row = in_rows[out_row];
    const int dst_sample_size = dst_counts[out_row];
    const int out_row_start = dst_ptr[out_row];
    for (int idx = threadIdx.x; idx < dst_sample_size; idx += WARP_SIZE) {
      dst_outputs[out_row_start + idx] = row;
    }
    out_row += BLOCK_WARPS;
  }
}

template <typename T, typename Context>
void GraphReindexKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& neighbors,
                        const DenseTensor& count,
                        const paddle::optional<DenseTensor>& hashtable_value,
                        const paddle::optional<DenseTensor>& hashtable_index,
                        bool flag_buffer_hashtable,
                        DenseTensor* reindex_src,
                        DenseTensor* reindex_dst,
                        DenseTensor* out_nodes) {
  const T* x_data = x.data<T>();
  const T* neighbors_data = neighbors.data<T>();
  const int* count_data = count.data<int>();
  const int bs = x.dims()[0];
  const int num_edges = neighbors.dims()[0];
  reindex_src->Resize({num_edges});

  T* reindex_src_data = dev_ctx.template Alloc<T>(reindex_src);
  thrust::device_ptr<T> src_outputs(reindex_src_data);

  thrust::device_vector<T> unique_nodes;
  thrust::copy(neighbors_data, neighbors_data + num_edges, src_outputs);

  if (flag_buffer_hashtable) {
    // Here we directly use buffer tensor to act as a hash table.
    DenseTensor hashtable_value_out(hashtable_value->type());
    const auto* ph_value = hashtable_value.get_ptr();
    hashtable_value_out.ShareDataWith(*ph_value);
    DenseTensor hashtable_index_out(hashtable_index->type());
    const auto* ph_index = hashtable_index.get_ptr();
    hashtable_index_out.ShareDataWith(*ph_index);
    int* hashtable_value_data =
        hashtable_value_out.mutable_data<int>(dev_ctx.GetPlace());
    int* hashtable_index_data =
        hashtable_index_out.mutable_data<int>(dev_ctx.GetPlace());
    BufferReindex<T, Context>(dev_ctx,
                              x_data,
                              src_outputs,
                              &unique_nodes,
                              bs,
                              hashtable_value_data,
                              hashtable_index_data,
                              num_edges);
  } else {
    Reindex<T, Context>(
        dev_ctx, x_data, src_outputs, &unique_nodes, bs, num_edges);
  }

  // Get reindex dst edge.
  // Add support for multi-type edges reindex.
  int num_ac_count = count.dims()[0];
  int num_edge_types = num_ac_count / bs;
  thrust::device_vector<int> unique_dst_reindex(bs);
  thrust::sequence(unique_dst_reindex.begin(), unique_dst_reindex.end());
  constexpr int BLOCK_WARPS = 128 / WARP_SIZE;
  constexpr int TILE_SIZE = BLOCK_WARPS * 16;
  const dim3 block(WARP_SIZE, BLOCK_WARPS);
  const dim3 grid((bs + TILE_SIZE - 1) / TILE_SIZE);
  reindex_dst->Resize({num_edges});
  T* reindex_dst_data = dev_ctx.template Alloc<T>(reindex_dst);
  int begin = 0;
  for (int i = 0; i < num_edge_types; i++) {
    thrust::device_vector<int> dst_ptr(bs);
    thrust::exclusive_scan(
        count_data + i * bs, count_data + (i + 1) * bs, dst_ptr.begin());

    GetDstEdgeCUDAKernel<T, BLOCK_WARPS, TILE_SIZE>
        <<<grid, block, 0, dev_ctx.stream()>>>(
            bs,
            thrust::raw_pointer_cast(unique_dst_reindex.data()),
            count_data + i * bs,
            thrust::raw_pointer_cast(dst_ptr.data()),
            reindex_dst_data + begin);

    int count_i =
        thrust::reduce(thrust::device_pointer_cast(count_data) + i * bs,
                       thrust::device_pointer_cast(count_data) + (i + 1) * bs);
    begin += count_i;
  }

  out_nodes->Resize({static_cast<int>(unique_nodes.size())});
  T* out_nodes_data = dev_ctx.template Alloc<T>(out_nodes);
  thrust::copy(unique_nodes.begin(), unique_nodes.end(), out_nodes_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    graph_reindex, GPU, ALL_LAYOUT, phi::GraphReindexKernel, int, int64_t) {}
