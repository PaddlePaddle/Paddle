#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/instance_norm_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/norm_utils.h"
#include "paddle/phi/kernels/gpu/instance_norm_utils.h"

namespace phi {
template <typename T, int BlockDim>
static __global__ void GradComputeDX(const T *dy,
                                     const BatchNormParamType<T> *scale,
                                     const BatchNormParamType<T> *mean,
                                     const T *x,
                                     const BatchNormParamType<T> *variance,
                                     const int C,
                                     const int sample_size,
                                     T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;
  BatchNormParamType<T> mean_val = mean[ncid];
  BatchNormParamType<T> inv_var_val = variance[ncid];
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;
  BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
  BatchNormParamType<T> dy_x_sub_mean_sum =
      static_cast<BatchNormParamType<T>>(0);

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    BatchNormParamType<T> dy_i = static_cast<BatchNormParamType<T>>(dy[i]);
    dy_sum += dy_i;
    dy_x_sub_mean_sum +=
        dy_i * (static_cast<BatchNormParamType<T>>(x[i]) - mean_val);
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_x_sub_mean_sum =
      BlockReduce(dy_x_sub_mean_storage).Reduce(dy_x_sub_mean_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
  }
  __syncthreads();
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    dx[i] =
        (static_cast<BatchNormParamType<T>>(dy[i]) -
         dy_sum_val / static_cast<BatchNormParamType<T>>(sample_size) -
         (static_cast<BatchNormParamType<T>>(x[i]) - mean_val) *
             dy_x_sub_mean_sum_val * inv_var_val * inv_var_val / sample_size) *
        scale[c] * inv_var_val;
  }
}

static __device__ __forceinline__ float real_sqrt(float x) {
  return 1. / sqrtf(x);
}
static __device__ __forceinline__ double real_sqrt(double x) {
  return 1. / sqrt(x);
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDX(const T *x,
                                    const T *mean,
                                    const T *variance,
                                    const T *ddx,
                                    const T *dy,
                                    const T *scale,
                                    const T *ddscale,
                                    int C,
                                    int sample_size,
                                    const double epsilon,
                                    T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  T mean_val = mean[ncid];
  T var_val = variance[ncid];

  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ T dy_sum_val;
  __shared__ T ddx_sum_val;
  __shared__ T dy_mul_ddx_sum_val;
  __shared__ T dy_mul_x_sub_mean_sum_val;
  __shared__ T ddx_mul_x_sub_mean_sum_val;

  T dy_sum = 0;
  T ddx_sum = 0;
  T dy_mul_ddx_sum = 0;
  T dy_mul_x_sub_mean_sum = 0;
  T ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T ddx_i = ddx[i];
    T dy_i = dy[i];
    T tmp = x[i] - mean_val;

    dy_sum += dy_i;
    ddx_sum += ddx_i;
    dy_mul_ddx_sum += (ddx_i * dy_i);

    dy_mul_x_sub_mean_sum += (dy_i * tmp);
    ddx_mul_x_sub_mean_sum += (ddx_i * tmp);
  }

  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  dy_mul_ddx_sum =
      BlockReduce(dy_mul_ddx_storage).Reduce(dy_mul_ddx_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    ddx_sum_val = ddx_sum;
    dy_mul_ddx_sum_val = dy_mul_ddx_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dx[i] +=
          ((x[i] - mean_val) * var_val * var_val * var_val / sample_size *
               (ddx_sum_val * dy_sum_val / sample_size - dy_mul_ddx_sum_val +
                3. * dy_mul_x_sub_mean_sum_val * var_val *
                    ddx_mul_x_sub_mean_sum_val * var_val / sample_size) +
           ddx_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
               var_val * (dy_sum_val / sample_size - dy[i]) +
           dy_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
               var_val * (ddx_sum_val / sample_size - ddx[i])) *
          scale[c];
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dx[i] += (dy[i] * var_val - dy_sum_val / sample_size * var_val -
                (x[i] - mean_val) * var_val * dy_mul_x_sub_mean_sum_val *
                    var_val / sample_size) *
               ddscale[c];
    }
  }
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDDY(const T *x,
                                     const T *mean,
                                     const T *variance,
                                     const T *ddscale,
                                     const T *ddbias,
                                     const T *ddx,
                                     const T *scale,
                                     int C,
                                     int sample_size,
                                     const double epsilon,
                                     T *ddy) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;
  T mean_val = mean[ncid];
  T var_val = variance[ncid];
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ T ddx_sum_val;
  __shared__ T ddx_mul_x_sub_mean_sum_val;

  T ddx_sum = 0;
  T ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T ddx_i = ddx[i];
    ddx_sum += ddx_i;
    ddx_mul_x_sub_mean_sum += (ddx_i * (x[i] - mean_val));
  }
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    ddx_sum_val = ddx_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();
  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += scale[c] * var_val *
                (ddx[i] - ddx_sum_val / sample_size -
                 (x[i] - mean_val) * var_val * ddx_mul_x_sub_mean_sum_val *
                     var_val / sample_size);
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += (x[i] - mean_val) * var_val * ddscale[c];
    }
  }
  __syncthreads();
  if (ddbias != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] += ddbias[c];
    }
  }
}

template <typename T, int BlockDim>
__global__ void DoubleGradComputeDScale(const T *x,
                                        const T *mean,
                                        const T *variance,
                                        const T *ddx,
                                        const T *dy,
                                        int C,
                                        int sample_size,
                                        const double epsilon,
                                        T *dscale) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;
  T mean_val = mean[ncid];
  T var_val = variance[ncid];
  typedef hipcub::BlockReduce<T, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage dscale_tmp_storage;
  __shared__ T dy_sum_val;
  __shared__ T dy_mul_x_sub_mean_sum_val;

  T dy_sum = 0;
  T dy_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T dy_i = dy[i];
    dy_sum += dy_i;
    dy_mul_x_sub_mean_sum += (dy_i * (x[i] - mean_val));
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
  }
  __syncthreads();
  if (ddx != nullptr) {
    T dscale_tmp = 0;
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dscale_tmp += ddx[i] * var_val *
                    (dy[i] - dy_sum_val / sample_size -
                     dy_mul_x_sub_mean_sum_val * (x[i] - mean_val) * var_val *
                         var_val / sample_size);
    }
    dscale_tmp = BlockReduce(dscale_tmp_storage).Reduce(dscale_tmp, hipcub::Sum());
    if (threadIdx.x == 0) {
      dscale[ncid] += dscale_tmp;
    }
    __syncthreads();
  }
}

template <typename T, typename Context>
void InstanceNormGradKernel(const Context &dev_ctx,
                            const DenseTensor &x,
                            const paddle::optional<DenseTensor> &scale,
                            const DenseTensor &saved_mean,
                            const DenseTensor &saved_variance,
                            const DenseTensor &d_y,
                            float epsilon_f,
                            DenseTensor *d_x,
                            DenseTensor *d_scale,
                            DenseTensor *d_bias) {
  double epsilon = static_cast<double>(epsilon_f);
  const auto *scale_ptr = scale.get_ptr();

  const auto &x_dims = x.dims();

  int N, C, H, W, D;
  funcs::ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
  int NxC = N * C;

  DenseTensor x_tmp, d_y_tmp;
  x_tmp.ShareDataWith(x).Resize({1, NxC, H, W, D});
  d_y_tmp.ShareDataWith(d_y).Resize({1, NxC, H, W, D});

  dev_ctx.template Alloc<T>(d_x);
  if (d_scale && d_bias) {
    dev_ctx.template Alloc<T>(d_scale);
    dev_ctx.template Alloc<T>(d_bias);
  }
  if (scale_ptr) {
    PADDLE_ENFORCE_EQ(
        scale_ptr->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The `shape` in InstanceNormOp is invalid: "
            "the size of scale's dimensions must be equal to 1. But "
            "received: the size of scale's dimensions"
            "is [%d]",
            scale_ptr->dims().size()));
    PADDLE_ENFORCE_EQ(scale_ptr->dims()[0],
                      C,
                      phi::errors::InvalidArgument(
                          "The `shape` in InstanceNormOp is invalid: "
                          "the first dimension of scale must be equal to "
                          "Channels([%d]). But received: "
                          "the first dimension of scale is [%d],"
                          "the dimensions of scale is [%s], ",
                          C,
                          scale_ptr->dims()[0],
                          scale_ptr->dims()));
  }

  phi::funcs::SetConstant<GPUContext, T> set_constant;

  const int n = x.numel();
  const int block = 512;
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(max_threads / block, 1);
  const int grid = std::min(NxC, max_blocks);
  const int grid1 = (C + block - 1) / block;

  DenseTensor scale_tmp;
  scale_tmp.Resize({NxC});
  dev_ctx.template Alloc<T>(&scale_tmp);

  DenseTensor d_scale_tmp;
  d_scale_tmp.Resize({NxC});
  dev_ctx.template Alloc<T>(&d_scale_tmp);

  DenseTensor d_bias_tmp;
  d_bias_tmp.Resize({NxC});
  dev_ctx.template Alloc<T>(&d_bias_tmp);

  if (scale_ptr) {
    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        scale_ptr->data<T>(), scale_tmp.data<T>(), N, C);
  } else {
    set_constant(dev_ctx, &scale_tmp, static_cast<T>(1));
  }

  std::vector<int> dims;
  std::vector<int> strides;
  dims = {1, NxC, H, W, D};
  strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

  if ((H * W * D) == 1) {
    phi::Copy(dev_ctx, d_y, dev_ctx.GetPlace(), false, d_x);
    phi::funcs::SetConstant<GPUContext, BatchNormParamType<T>> functor;
    functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
    functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
    return;
  }

#ifdef PADDLE_WITH_HIP
  miopenTensorDescriptor_t data_desc_;
  miopenTensorDescriptor_t in_param_desc_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenCreateTensorDescriptor(&in_param_desc_));
#else
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t in_param_desc_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));
#endif

  if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
    LOG(ERROR) << "Provided epsilon is smaller than "
               << "HIPDNN_BN_MIN_EPSILON. Setting it to "
               << "HIPDNN_BN_MIN_EPSILON instead.";
  }
  epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenSetTensorDescriptor(
          data_desc_,
          CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4,
          const_cast<int *>(dims.data()),
          const_cast<int *>(strides.data())));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenDeriveBNTensorDescriptor(
          in_param_desc_, data_desc_, miopenBNSpatial));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_,
          CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4,
          dims.data(),
          strides.data()));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDeriveBNTensorDescriptor(
          in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));
#endif

  const auto *saved_mean_data =
      saved_mean.template data<BatchNormParamType<T>>();
  const auto *saved_var_data =
      saved_variance.template data<BatchNormParamType<T>>();
  if (d_scale && d_bias) {
#ifdef PADDLE_WITH_HIP
    PADDLE_ENFORCE_GPU_SUCCESS(
        paddle::platform::dynload::miopenBatchNormalizationBackward(
            dev_ctx.cudnn_handle(),
            miopenBNSpatial,
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            data_desc_,
            x_tmp.template data<T>(),
            data_desc_,
            d_y_tmp.template data<T>(),
            data_desc_,
            d_x->template data<T>(),
            in_param_desc_,
            scale_tmp.template data<BatchNormParamType<T>>(),
            d_scale_tmp.template data<BatchNormParamType<T>>(),
            d_bias_tmp.template data<BatchNormParamType<T>>(),
            epsilon,
            saved_mean_data,
            saved_var_data));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(
        paddle::platform::dynload::hipdnnBatchNormalizationBackward(
            dev_ctx.cudnn_handle(),
            HIPDNN_BATCHNORM_SPATIAL,
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            data_desc_,
            x_tmp.template data<T>(),
            data_desc_,
            d_y_tmp.template data<T>(),
            data_desc_,
            d_x->template data<T>(),
            in_param_desc_,
            scale_tmp.template data<BatchNormParamType<T>>(),
            d_scale_tmp.template data<BatchNormParamType<T>>(),
            d_bias_tmp.template data<BatchNormParamType<T>>(),
            epsilon,
            saved_mean_data,
            saved_var_data));
#endif
  } else {
    if (d_x) {
      GradComputeDX<T, block><<<NxC, block, 0, dev_ctx.stream()>>>(
          d_y.data<T>(),
          scale_tmp.data<BatchNormParamType<T>>(),
          saved_mean_data,
          x.data<T>(),
          saved_var_data,
          C,
          H * W * D,
          d_x->data<T>());
    }
  }

  if (d_scale && d_bias) {
    add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        d_scale_tmp.data<T>(), d_scale->data<T>(), N, C);
    add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        d_bias_tmp.data<T>(), d_bias->data<T>(), N, C);
  }

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenDestroyTensorDescriptor(in_param_desc_));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
#endif
}

template <typename T, typename Context>
void InstanceNormDoubleGradKernel(const Context &dev_ctx,
                                  const DenseTensor &x,
                                  const paddle::optional<DenseTensor> &scale,
                                  const DenseTensor &saved_mean,
                                  const DenseTensor &saved_variance,
                                  const DenseTensor &dy,
                                  const paddle::optional<DenseTensor> &ddx,
                                  const paddle::optional<DenseTensor> &ddscale,
                                  const paddle::optional<DenseTensor> &ddbias,
                                  float epsilon_f,
                                  DenseTensor *dx,
                                  DenseTensor *dscale,
                                  DenseTensor *ddy) {
  const auto *Scale = scale.get_ptr();
  const auto *ddX = ddx.get_ptr();
  const auto *ddScale = ddscale.get_ptr();
  const auto *ddBias = ddbias.get_ptr();
  const double epsilon = static_cast<double>(epsilon_f);
  const T *x_data = x.data<T>();
  const T *dy_data = dy.data<T>();
  const T *ddx_data = (ddX == nullptr ? nullptr : ddX->data<T>());
  const T *ddscale_data = (ddScale == nullptr ? nullptr : ddScale->data<T>());
  const T *ddbias_data = (ddScale == nullptr ? nullptr : ddBias->data<T>());
  const T *mean_data = saved_mean.data<T>();
  const T *variance_data = saved_variance.data<T>();
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  auto &x_dims = x.dims();
  int N, C, H, W, D;
  funcs::ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
  int NxC = N * C;
  const int n = x.numel();
  int sample_size = n / N / C;

  DenseTensor scale_tmp;
  if (!Scale) {
    scale_tmp.Resize({C});
    dev_ctx.template Alloc<T>(&scale_tmp);
    set_zero(dev_ctx, &scale_tmp, static_cast<T>(1));
  }
  const T *scale_data = Scale ? Scale->data<T>() : scale_tmp.data<T>();
  const int block = 512;
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(max_threads / block, 1);
  const int grid = NxC;
  const int grid1 = (C + block - 1) / block;

  if (dx) {
    T *dx_data = dev_ctx.template Alloc<T>(dx);
    set_zero(dev_ctx, dx, static_cast<T>(0));
    DoubleGradComputeDX<T, block>
        <<<grid, block, 0, dev_ctx.stream()>>>(x_data,
                                               mean_data,
                                               variance_data,
                                               ddx_data,
                                               dy_data,
                                               scale_data,
                                               ddscale_data,
                                               C,
                                               sample_size,
                                               epsilon,
                                               dx_data);
  }
  if (dscale) {
    DenseTensor dscale_tmp;
    dscale_tmp.Resize({NxC});
    dev_ctx.template Alloc<T>(&dscale_tmp);
    set_zero(dev_ctx, &dscale_tmp, static_cast<T>(0));
    T *dscale_tmp_data = dscale_tmp.data<T>();

    T *dscale_data = dev_ctx.template Alloc<T>(dscale);
    set_zero(dev_ctx, dscale, static_cast<T>(0));
    DoubleGradComputeDScale<T, block>
        <<<grid, block, 0, dev_ctx.stream()>>>(x_data,
                                               mean_data,
                                               variance_data,
                                               ddx_data,
                                               dy_data,
                                               C,
                                               sample_size,
                                               epsilon,
                                               dscale_tmp_data);
    add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        dscale_tmp.data<T>(), dscale->data<T>(), N, C);
  }
  if (ddy) {
    T *ddy_data = dev_ctx.template Alloc<T>(ddy);
    set_zero(dev_ctx, ddy, static_cast<T>(0));
    DoubleGradComputeDDY<T, block>
        <<<grid, block, 0, dev_ctx.stream()>>>(x_data,
                                               mean_data,
                                               variance_data,
                                               ddscale_data,
                                               ddbias_data,
                                               ddx_data,
                                               scale_data,
                                               C,
                                               sample_size,
                                               epsilon,
                                               ddy_data);
  }
}
}  // namespace phi

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(
    instance_norm_grad, GPU, ALL_LAYOUT, phi::InstanceNormGradKernel, float) {}
PD_REGISTER_KERNEL(instance_norm_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormDoubleGradKernel,
                   float) {}
#else
PD_REGISTER_KERNEL(instance_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormGradKernel,
                   float,
                   double) {}
PD_REGISTER_KERNEL(instance_norm_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormDoubleGradKernel,
                   float,
                   double) {}
#endif
