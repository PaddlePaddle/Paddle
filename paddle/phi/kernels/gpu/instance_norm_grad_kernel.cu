#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/instance_norm_grad_kernel.h"

#include "paddle/fluid/operators/norm_utils.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/copy_kernel.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/instance_norm_utils.h"

namespace phi {

template <typename T, int BlockDim>
static __global__ void GradComputeDX(const T *dy,
                                     const BatchNormParamType<T> *scale,
                                     const BatchNormParamType<T> *mean,
                                     const T *x,
                                     const BatchNormParamType<T> *variance,
                                     const int C,
                                     const int sample_size,
                                     T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  BatchNormParamType<T> mean_val = mean[ncid];
  BatchNormParamType<T> inv_var_val = variance[ncid];

  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;

  BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
  BatchNormParamType<T> dy_x_sub_mean_sum =
      static_cast<BatchNormParamType<T>>(0);

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    BatchNormParamType<T> dy_i = static_cast<BatchNormParamType<T>>(dy[i]);
    dy_sum += dy_i;
    dy_x_sub_mean_sum +=
        dy_i * (static_cast<BatchNormParamType<T>>(x[i]) - mean_val);
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_x_sub_mean_sum =
      BlockReduce(dy_x_sub_mean_storage).Reduce(dy_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
  }
  __syncthreads();

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    dx[i] =
        (static_cast<BatchNormParamType<T>>(dy[i]) -
         dy_sum_val / static_cast<BatchNormParamType<T>>(sample_size) -
         (static_cast<BatchNormParamType<T>>(x[i]) - mean_val) *
             dy_x_sub_mean_sum_val * inv_var_val * inv_var_val / sample_size) *
        scale[c] * inv_var_val;
  }
}

template <typename T, typename Context>
void InstanceNormGradKernel(const Context &dev_ctx,
                            const DenseTensor &x,
                            const DenseTensor &d_y,
                            const paddle::optional<DenseTensor> &scale,
                            const DenseTensor &saved_mean,
                            const DenseTensor &saved_variance,
                            float epsilon_f,
                            DenseTensor *d_x,
                            DenseTensor *d_scale,
                            DenseTensor *d_bias) {
  double epsilon = static_cast<double>(epsilon_f);
  const auto *scale_ptr = scale.get_ptr();

  const auto &x_dims = x.dims();

  int N, C, H, W, D;
  paddle::operators::ExtractNCWHD(
      x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
  int NxC = N * C;

  DenseTensor x_tmp, d_y_tmp;
  x_tmp.ShareDataWith(x).Resize({1, NxC, H, W, D});
  d_y_tmp.ShareDataWith(d_y).Resize({1, NxC, H, W, D});

  dev_ctx.template Alloc<T>(d_x);
  if (d_scale && d_bias) {
    dev_ctx.template Alloc<T>(d_scale);
    dev_ctx.template Alloc<T>(d_bias);
  }
  if (scale_ptr) {
    PADDLE_ENFORCE_EQ(
        scale_ptr->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The `shape` in InstanceNormOp is invalid: "
            "the size of scale's dimensions must be equal to 1. But "
            "received: the size of scale's dimensions"
            "is [%d]",
            scale_ptr->dims().size()));
    PADDLE_ENFORCE_EQ(scale_ptr->dims()[0],
                      C,
                      phi::errors::InvalidArgument(
                          "The `shape` in InstanceNormOp is invalid: "
                          "the first dimension of scale must be equal to "
                          "Channels([%d]). But received: "
                          "the first dimension of scale is [%d],"
                          "the dimensions of scale is [%s], ",
                          C,
                          scale_ptr->dims()[0],
                          scale_ptr->dims()));
  }

  phi::funcs::SetConstant<GPUContext, T> set_constant;

  const int n = x.numel();
  const int block = 512;
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(max_threads / block, 1);
  const int grid = std::min(NxC, max_blocks);
  const int grid1 = (C + block - 1) / block;

  DenseTensor scale_tmp;
  scale_tmp.Resize({NxC});
  dev_ctx.template Alloc<T>(&scale_tmp);

  DenseTensor d_scale_tmp;
  d_scale_tmp.Resize({NxC});
  dev_ctx.template Alloc<T>(&d_scale_tmp);

  DenseTensor d_bias_tmp;
  d_bias_tmp.Resize({NxC});
  dev_ctx.template Alloc<T>(&d_bias_tmp);

  if (scale_ptr) {
    repeat_param<T><<<grid, block, 0, dev_ctx.stream()>>>(
        scale_ptr->data<T>(), scale_tmp.data<T>(), N, C);
  } else {
    set_constant(dev_ctx, &scale_tmp, static_cast<T>(1));
  }

  std::vector<int> dims;
  std::vector<int> strides;
  dims = {1, NxC, H, W, D};
  strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

  if ((H * W * D) == 1) {
    phi::Copy(dev_ctx, d_y, dev_ctx.GetPlace(), false, d_x);
    phi::funcs::SetConstant<GPUContext, BatchNormParamType<T>> functor;
    functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
    functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
    return;
  }

#ifdef PADDLE_WITH_HIP
  miopenTensorDescriptor_t data_desc_;
  miopenTensorDescriptor_t in_param_desc_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenCreateTensorDescriptor(&in_param_desc_));
#else
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t in_param_desc_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));
#endif

  if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
    LOG(ERROR) << "Provided epsilon is smaller than "
               << "HIPDNN_BN_MIN_EPSILON. Setting it to "
               << "HIPDNN_BN_MIN_EPSILON instead.";
  }
  epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenSetTensorDescriptor(
          data_desc_,
          CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4,
          const_cast<int *>(dims.data()),
          const_cast<int *>(strides.data())));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenDeriveBNTensorDescriptor(
          in_param_desc_, data_desc_, miopenBNSpatial));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_,
          CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4,
          dims.data(),
          strides.data()));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDeriveBNTensorDescriptor(
          in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));
#endif

  const auto *saved_mean_data =
      saved_mean.template data<BatchNormParamType<T>>();
  const auto *saved_var_data =
      saved_variance.template data<BatchNormParamType<T>>();
  if (d_scale && d_bias) {
#ifdef PADDLE_WITH_HIP
    PADDLE_ENFORCE_GPU_SUCCESS(
        paddle::platform::dynload::miopenBatchNormalizationBackward(
            dev_ctx.cudnn_handle(),
            miopenBNSpatial,
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            data_desc_,
            x_tmp.template data<T>(),
            data_desc_,
            d_y_tmp.template data<T>(),
            data_desc_,
            d_x->template data<T>(),
            in_param_desc_,
            scale_tmp.template data<BatchNormParamType<T>>(),
            d_scale_tmp.template data<BatchNormParamType<T>>(),
            d_bias_tmp.template data<BatchNormParamType<T>>(),
            epsilon,
            saved_mean_data,
            saved_var_data));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(
        paddle::platform::dynload::hipdnnBatchNormalizationBackward(
            dev_ctx.cudnn_handle(),
            HIPDNN_BATCHNORM_SPATIAL,
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            data_desc_,
            x_tmp.template data<T>(),
            data_desc_,
            d_y_tmp.template data<T>(),
            data_desc_,
            d_x->template data<T>(),
            in_param_desc_,
            scale_tmp.template data<BatchNormParamType<T>>(),
            d_scale_tmp.template data<BatchNormParamType<T>>(),
            d_bias_tmp.template data<BatchNormParamType<T>>(),
            epsilon,
            saved_mean_data,
            saved_var_data));
#endif
  } else {
    if (d_x) {
      GradComputeDX<T, block><<<NxC, block, 0, dev_ctx.stream()>>>(
          d_y.data<T>(),
          scale_tmp.data<BatchNormParamType<T>>(),
          saved_mean_data,
          x.data<T>(),
          saved_var_data,
          C,
          H * W * D,
          d_x->data<T>());
    }
  }

  if (d_scale && d_bias) {
    add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        d_scale_tmp.data<T>(), d_scale->data<T>(), N, C);
    add_param<T, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        d_bias_tmp.data<T>(), d_bias->data<T>(), N, C);
  }

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::miopenDestroyTensorDescriptor(in_param_desc_));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
#endif
}
}  // namespace phi

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(
    instance_norm_grad, GPU, ALL_LAYOUT, phi::InstanceNormGradKernel, float) {}
#else
PD_REGISTER_KERNEL(instance_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormGradKernel,
                   float,
                   double) {}
#endif
