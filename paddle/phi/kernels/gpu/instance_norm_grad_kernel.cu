#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/instance_norm_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/norm_utils.h"
#include "paddle/phi/kernels/gpu/instance_norm_utils.h"

#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/device_context.h"

namespace phi {
template <typename T, typename AccT, int BlockDim>
static __global__ void GradComputeDX(const T *dy,
                                     const BatchNormParamType<T> *scale,
                                     const BatchNormParamType<T> *mean,
                                     const T *x,
                                     const BatchNormParamType<T> *variance,
                                     const int C,
                                     const int sample_size,
                                     T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;
  BatchNormParamType<T> mean_val = mean[ncid];
  BatchNormParamType<T> inv_var_val = variance[ncid];
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_x_sub_mean_storage;
  __shared__ BatchNormParamType<T> dy_sum_val;
  __shared__ BatchNormParamType<T> dy_x_sub_mean_sum_val;
  BatchNormParamType<T> dy_sum = static_cast<BatchNormParamType<T>>(0);
  BatchNormParamType<T> dy_x_sub_mean_sum =
      static_cast<BatchNormParamType<T>>(0);

  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    BatchNormParamType<T> dy_i = static_cast<BatchNormParamType<T>>(dy[i]);
    dy_sum += dy_i;
    dy_x_sub_mean_sum +=
        dy_i * (static_cast<BatchNormParamType<T>>(x[i]) - mean_val);
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_x_sub_mean_sum =
      BlockReduce(dy_x_sub_mean_storage).Reduce(dy_x_sub_mean_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_x_sub_mean_sum_val = dy_x_sub_mean_sum;
  }
  __syncthreads();
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    AccT tmp =
        (static_cast<BatchNormParamType<T>>(dy[i]) -
         dy_sum_val / static_cast<BatchNormParamType<T>>(sample_size) -
         (static_cast<BatchNormParamType<T>>(x[i]) - mean_val) *
             dy_x_sub_mean_sum_val * inv_var_val * inv_var_val / sample_size) *
        static_cast<BatchNormParamType<T>>(scale[c]) * inv_var_val;
    dx[i] = static_cast<T>(dx[i]);
  }
}

static __device__ __forceinline__ float real_sqrt(float x) {
  return 1. / sqrtf(x);
}
static __device__ __forceinline__ double real_sqrt(double x) {
  return 1. / sqrt(x);
}

template <typename T, typename AccT, int BlockDim>
__global__ void DoubleGradComputeDX(const T *x,
                                    const AccT *mean,
                                    const AccT *variance,
                                    const T *ddx,
                                    const T *dy,
                                    const AccT *scale,
                                    const AccT *ddscale,
                                    int C,
                                    int sample_size,
                                    const double epsilon,
                                    T *dx) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;

  AccT mean_val = mean[ncid];
  AccT var_val = variance[ncid];

  typedef hipcub::BlockReduce<AccT, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_ddx_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ AccT dy_sum_val;
  __shared__ AccT ddx_sum_val;
  __shared__ AccT dy_mul_ddx_sum_val;
  __shared__ AccT dy_mul_x_sub_mean_sum_val;
  __shared__ AccT ddx_mul_x_sub_mean_sum_val;

  AccT dy_sum = 0;
  AccT ddx_sum = 0;
  AccT dy_mul_ddx_sum = 0;
  AccT dy_mul_x_sub_mean_sum = 0;
  AccT ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    AccT ddx_i = static_cast<AccT>(ddx[i]);
    AccT dy_i = static_cast<AccT>(dy[i]);
    AccT tmp = static_cast<AccT>(x[i]) - mean_val;

    dy_sum += dy_i;
    ddx_sum += ddx_i;
    dy_mul_ddx_sum += (ddx_i * dy_i);

    dy_mul_x_sub_mean_sum += dy_i * tmp;
    ddx_mul_x_sub_mean_sum += ddx_i * tmp;
  }

  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  dy_mul_ddx_sum =
      BlockReduce(dy_mul_ddx_storage).Reduce(dy_mul_ddx_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    ddx_sum_val = ddx_sum;
    dy_mul_ddx_sum_val = dy_mul_ddx_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();

  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      AccT tmp = static_cast<AccT>(dx[i]);
      tmp += (static_cast<AccT>(x[i]) - mean_val) * var_val * var_val *
                 var_val / sample_size *
                 (ddx_sum_val * dy_sum_val / sample_size - dy_mul_ddx_sum_val +
                  3. * dy_mul_x_sub_mean_sum_val * var_val *
                      ddx_mul_x_sub_mean_sum_val * var_val / sample_size) +
             ddx_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
                 var_val *
                 (dy_sum_val / sample_size - static_cast<AccT>(dy[i]) +
                  dy_mul_x_sub_mean_sum_val * var_val / sample_size * var_val *
                      var_val *
                      (ddx_sum_val / sample_size - static_cast<AccT>(ddx[i]))) *
                 scale[c];
      dx[i] = static_cast<T>(tmp);
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      AccT tmp = static_cast<AccT>(dx[i]);
      tmp += (static_cast<AccT>(dy[i]) * var_val -
              dy_sum_val / sample_size * var_val -
              (static_cast<AccT>(x[i]) - mean_val) * var_val *
                  dy_mul_x_sub_mean_sum_val * var_val / sample_size) *
             ddscale[c];
      dx[i] = static_cast<T>(tmp);
    }
  }
}

template <typename T, typename AccT, int BlockDim>
__global__ void DoubleGradComputeDDY(const T *x,
                                     const AccT *mean,
                                     const AccT *variance,
                                     const AccT *ddscale,
                                     const AccT *ddbias,
                                     const T *ddx,
                                     const AccT *scale,
                                     int C,
                                     int sample_size,
                                     const double epsilon,
                                     T *ddy) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;
  AccT mean_val = mean[ncid];
  AccT var_val = variance[ncid];
  typedef hipcub::BlockReduce<AccT, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage ddx_storage;
  __shared__ typename BlockReduce::TempStorage ddx_mul_x_sub_mean_storage;
  __shared__ AccT ddx_sum_val;
  __shared__ AccT ddx_mul_x_sub_mean_sum_val;

  AccT ddx_sum = 0;
  AccT ddx_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    AccT ddx_i = static_cast<AccT>(ddx[i]);
    ddx_sum = ddx_i;
    ddx_mul_x_sub_mean_sum += (ddx_i * (static_cast<AccT>(x[i]) - mean_val));
  }
  ddx_sum = BlockReduce(ddx_storage).Reduce(ddx_sum, hipcub::Sum());
  ddx_mul_x_sub_mean_sum = BlockReduce(ddx_mul_x_sub_mean_storage)
                               .Reduce(ddx_mul_x_sub_mean_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    ddx_sum_val = ddx_sum;
    ddx_mul_x_sub_mean_sum_val = ddx_mul_x_sub_mean_sum;
  }
  __syncthreads();
  if (ddx != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      AccT tmp = static_cast<AccT>(ddy[i]);
      tmp += static_cast<AccT>(ddy[i]) +
             scale[c] * var_val *
                 (static_cast<AccT>(ddx[i]) - ddx_sum_val / sample_size -
                  (static_cast<AccT>(x[i]) - mean_val) * var_val *
                      ddx_mul_x_sub_mean_sum_val * var_val / sample_size);
      ddy[i] = static_cast<T>(tmp);
    }
  }
  __syncthreads();
  if (ddscale != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      AccT tmp = static_cast<AccT>(ddy[i]);
      tmp += (static_cast<AccT>(x[i]) - mean_val) * var_val * ddscale[c];
      ddy[i] = static_cast<T>(tmp);
    }
  }
  __syncthreads();
  if (ddbias != nullptr) {
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      ddy[i] = static_cast<T>(static_cast<AccT>(ddy[i]) + ddbias[c]);
    }
  }
}

template <typename T, typename AccT, int BlockDim>
__global__ void DoubleGradComputeDScale(const T *x,
                                        const AccT *mean,
                                        const AccT *variance,
                                        const T *ddx,
                                        const T *dy,
                                        int C,
                                        int sample_size,
                                        const double epsilon,
                                        AccT *dscale) {
  int beg_idx = blockIdx.x * sample_size + threadIdx.x;
  int end_idx = (blockIdx.x + 1) * sample_size;
  int ncid = blockIdx.x;
  int c = ncid % C;
  AccT mean_val = mean[ncid];
  AccT var_val = variance[ncid];
  typedef hipcub::BlockReduce<AccT, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage dy_storage;
  __shared__ typename BlockReduce::TempStorage dy_mul_x_sub_mean_storage;
  __shared__ typename BlockReduce::TempStorage dscale_tmp_storage;
  __shared__ AccT dy_sum_val;
  __shared__ AccT dy_mul_x_sub_mean_sum_val;

  AccT dy_sum = 0;
  AccT dy_mul_x_sub_mean_sum = 0;
  for (int i = beg_idx; i < end_idx; i += BlockDim) {
    T dy_i = dy[i];
    dy_sum = dy_sum + static_cast<AccT>(dy_i);
    dy_mul_x_sub_mean_sum =
        dy_mul_x_sub_mean_sum +
        (static_cast<AccT>(dy_i) * (static_cast<AccT>(x[i]) - mean_val));
  }
  dy_sum = BlockReduce(dy_storage).Reduce(dy_sum, hipcub::Sum());
  dy_mul_x_sub_mean_sum = BlockReduce(dy_mul_x_sub_mean_storage)
                              .Reduce(dy_mul_x_sub_mean_sum, hipcub::Sum());

  if (threadIdx.x == 0) {
    dy_sum_val = dy_sum;
    dy_mul_x_sub_mean_sum_val = dy_mul_x_sub_mean_sum;
  }
  __syncthreads();
  if (ddx != nullptr) {
    AccT dscale_tmp = 0;
    for (int i = beg_idx; i < end_idx; i += BlockDim) {
      dscale_tmp = dscale_tmp +
                   static_cast<AccT>(ddx[i]) * var_val *
                       (static_cast<AccT>(dy[i]) - dy_sum_val / sample_size -
                        dy_mul_x_sub_mean_sum_val *
                            (static_cast<AccT>(x[i]) - mean_val) * var_val *
                            var_val / sample_size);
    }
    dscale_tmp = BlockReduce(dscale_tmp_storage).Reduce(dscale_tmp, hipcub::Sum());
    if (threadIdx.x == 0) {
      dscale[ncid] = dscale[ncid] + dscale_tmp;
    }
    __syncthreads();
  }
}

template <typename T, typename Context>
void InstanceNormGradKernel(const Context &dev_ctx,
                            const DenseTensor &x,
                            const paddle::optional<DenseTensor> &scale,
                            const DenseTensor &saved_mean,
                            const DenseTensor &saved_variance,
                            const DenseTensor &d_y,
                            float epsilon_f,
                            DenseTensor *d_x,
                            DenseTensor *d_scale,
                            DenseTensor *d_bias) {
  using AccT = typename phi::dtype::MPTypeTrait<T>::Type;
  double epsilon = static_cast<double>(epsilon_f);
  const auto *scale_ptr = scale.get_ptr();

  const auto &x_dims = x.dims();

  int N, C, H, W, D;
  funcs::ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
  int NxC = N * C;

  DenseTensor x_tmp, d_y_tmp;
  x_tmp.ShareDataWith(x).Resize({1, NxC, H, W, D});
  d_y_tmp.ShareDataWith(d_y).Resize({1, NxC, H, W, D});

  dev_ctx.template Alloc<T>(d_x);
  if (d_scale && d_bias) {
    dev_ctx.template Alloc<AccT>(d_scale);
    dev_ctx.template Alloc<AccT>(d_bias);
  }
  if (scale_ptr) {
    PADDLE_ENFORCE_EQ(
        scale_ptr->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The `shape` in InstanceNormOp is invalid: "
            "the size of scale's dimensions must be equal to 1. But "
            "received: the size of scale's dimensions"
            "is [%d]",
            scale_ptr->dims().size()));
    PADDLE_ENFORCE_EQ(scale_ptr->dims()[0],
                      C,
                      phi::errors::InvalidArgument(
                          "The `shape` in InstanceNormOp is invalid: "
                          "the first dimension of scale must be equal to "
                          "Channels([%d]). But received: "
                          "the first dimension of scale is [%d],"
                          "the dimensions of scale is [%s], ",
                          C,
                          scale_ptr->dims()[0],
                          scale_ptr->dims()));
  }

  phi::funcs::SetConstant<GPUContext, AccT> set_constant;

  const int n = x.numel();
  const int block = 512;
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(max_threads / block, 1);
  const int grid = std::min(NxC, max_blocks);
  const int grid1 = (C + block - 1) / block;

  DenseTensor scale_tmp;
  scale_tmp.Resize({NxC});
  dev_ctx.template Alloc<AccT>(&scale_tmp);

  DenseTensor d_scale_tmp;
  d_scale_tmp.Resize({NxC});
  dev_ctx.template Alloc<AccT>(&d_scale_tmp);

  DenseTensor d_bias_tmp;
  d_bias_tmp.Resize({NxC});
  dev_ctx.template Alloc<AccT>(&d_bias_tmp);
  if (scale_ptr) {
    repeat_param<AccT><<<grid, block, 0, dev_ctx.stream()>>>(
        scale_ptr->data<AccT>(), scale_tmp.data<AccT>(), N, C);
  } else {
    set_constant(dev_ctx, &scale_tmp, static_cast<AccT>(1));
  }
  std::vector<int> dims;
  std::vector<int> strides;
  dims = {1, NxC, H, W, D};
  strides = {NxC * H * W * D, H * W * D, W * D, D, 1};

  if ((H * W * D) == 1) {
    phi::Copy(dev_ctx, d_y, dev_ctx.GetPlace(), false, d_x);
    phi::funcs::SetConstant<GPUContext, BatchNormParamType<T>> functor;
    functor(dev_ctx, d_scale, static_cast<BatchNormParamType<T>>(0));
    functor(dev_ctx, d_bias, static_cast<BatchNormParamType<T>>(0));
    return;
  }

#ifdef PADDLE_WITH_HIP
  miopenTensorDescriptor_t data_desc_;
  miopenTensorDescriptor_t in_param_desc_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::miopenCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::miopenCreateTensorDescriptor(&in_param_desc_));
#else
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t in_param_desc_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnCreateTensorDescriptor(&in_param_desc_));
#endif

  if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
    LOG(ERROR) << "Provided epsilon is smaller than "
               << "HIPDNN_BN_MIN_EPSILON. Setting it to "
               << "HIPDNN_BN_MIN_EPSILON instead.";
  }
  epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::miopenSetTensorDescriptor(
      data_desc_,
      CudnnDataType<T>::type,
      x_dims.size() > 3 ? x_dims.size() : 4,
      const_cast<int *>(dims.data()),
      const_cast<int *>(strides.data())));
  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::miopenDeriveBNTensorDescriptor(
      in_param_desc_, data_desc_, miopenBNSpatial));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipdnnSetTensorNdDescriptor(
      data_desc_,
      CudnnDataType<T>::type,
      x_dims.size() > 3 ? x_dims.size() : 4,
      dims.data(),
      strides.data()));
  PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipdnnDeriveBNTensorDescriptor(
      in_param_desc_, data_desc_, HIPDNN_BATCHNORM_SPATIAL));
#endif
  const auto *saved_mean_data =
      saved_mean.template data<BatchNormParamType<T>>();
  const auto *saved_var_data =
      saved_variance.template data<BatchNormParamType<T>>();

  if (d_scale && d_bias) {
#ifdef PADDLE_WITH_HIP
    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::miopenBatchNormalizationBackward(
        dev_ctx.cudnn_handle(),
        miopenBNSpatial,
        CudnnDataType<T>::kOne(),
        CudnnDataType<T>::kZero(),
        CudnnDataType<T>::kOne(),
        CudnnDataType<T>::kZero(),
        data_desc_,
        x_tmp.template data<T>(),
        data_desc_,
        d_y_tmp.template data<T>(),
        data_desc_,
        d_x->template data<T>(),
        in_param_desc_,
        scale_tmp.template data<BatchNormParamType<T>>(),
        d_scale_tmp.template data<BatchNormParamType<T>>(),
        d_bias_tmp.template data<BatchNormParamType<T>>(),
        epsilon,
        saved_mean_data,
        saved_var_data));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipdnnBatchNormalizationBackward(
        dev_ctx.cudnn_handle(),
        HIPDNN_BATCHNORM_SPATIAL,
        CudnnDataType<T>::kOne(),
        CudnnDataType<T>::kZero(),
        CudnnDataType<T>::kOne(),
        CudnnDataType<T>::kZero(),
        data_desc_,
        x_tmp.template data<T>(),
        data_desc_,
        d_y_tmp.template data<T>(),
        data_desc_,
        d_x->template data<T>(),
        in_param_desc_,
        scale_tmp.template data<BatchNormParamType<T>>(),
        d_scale_tmp.template data<BatchNormParamType<T>>(),
        d_bias_tmp.template data<BatchNormParamType<T>>(),
        epsilon,
        saved_mean_data,
        saved_var_data));
#endif
  } else {
    if (d_x) {
      GradComputeDX<T, AccT, block><<<NxC, block, 0, dev_ctx.stream()>>>(
          d_y.data<T>(),
          scale_tmp.data<BatchNormParamType<T>>(),
          saved_mean_data,
          x.data<T>(),
          saved_var_data,
          C,
          H * W * D,
          d_x->data<T>());
    }
  }
  if (d_scale && d_bias) {
    add_param<AccT, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        d_scale_tmp.data<AccT>(), d_scale->data<AccT>(), N, C);
    add_param<AccT, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        d_bias_tmp.data<AccT>(), d_bias->data<AccT>(), N, C);
  }

#ifdef PADDLE_WITH_HIP
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::miopenDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::miopenDestroyTensorDescriptor(in_param_desc_));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipdnnDestroyTensorDescriptor(in_param_desc_));
#endif
}

template <typename T, typename Context>
void InstanceNormDoubleGradKernel(const Context &dev_ctx,
                                  const DenseTensor &x,
                                  const paddle::optional<DenseTensor> &scale,
                                  const DenseTensor &saved_mean,
                                  const DenseTensor &saved_variance,
                                  const DenseTensor &dy,
                                  const paddle::optional<DenseTensor> &ddx,
                                  const paddle::optional<DenseTensor> &ddscale,
                                  const paddle::optional<DenseTensor> &ddbias,
                                  float epsilon_f,
                                  DenseTensor *dx,
                                  DenseTensor *dscale,
                                  DenseTensor *ddy) {
  using AccT = typename phi::dtype::MPTypeTrait<T>::Type;
  const auto *Scale = scale.get_ptr();
  const auto *ddX = ddx.get_ptr();
  const auto *ddScale = ddscale.get_ptr();
  const auto *ddBias = ddbias.get_ptr();
  const double epsilon = static_cast<double>(epsilon_f);
  const T *x_data = x.data<T>();
  const T *dy_data = dy.data<T>();
  const T *ddx_data = (ddX == nullptr ? nullptr : ddX->data<T>());
  const AccT *ddscale_data =
      (ddScale == nullptr ? nullptr : ddScale->data<AccT>());
  const AccT *ddbias_data =
      (ddScale == nullptr ? nullptr : ddBias->data<AccT>());
  const AccT *mean_data = saved_mean.data<AccT>();
  const AccT *variance_data = saved_variance.data<AccT>();
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  phi::funcs::SetConstant<GPUContext, AccT> set_zero_AccT;

  auto &x_dims = x.dims();
  int N, C, H, W, D;
  funcs::ExtractNCWHD(x_dims, DataLayout::kNCHW, &N, &C, &H, &W, &D);
  int NxC = N * C;
  const int n = x.numel();
  int sample_size = n / N / C;

  DenseTensor scale_tmp;
  if (!Scale) {
    scale_tmp.Resize({C});
    dev_ctx.template Alloc<AccT>(&scale_tmp);
    set_zero_AccT(dev_ctx, &scale_tmp, static_cast<AccT>(1));
  }
  const AccT *scale_data = Scale ? Scale->data<AccT>() : scale_tmp.data<AccT>();
  const int block = 512;
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(max_threads / block, 1);
  const int grid = NxC;
  const int grid1 = (C + block - 1) / block;

  if (dx) {
    T *dx_data = dev_ctx.template Alloc<T>(dx);
    set_zero(dev_ctx, dx, static_cast<T>(0));
    DoubleGradComputeDX<T, AccT, block>
        <<<grid, block, 0, dev_ctx.stream()>>>(x_data,
                                               mean_data,
                                               variance_data,
                                               ddx_data,
                                               dy_data,
                                               scale_data,
                                               ddscale_data,
                                               C,
                                               sample_size,
                                               epsilon,
                                               dx_data);
  }
  if (dscale) {
    DenseTensor dscale_tmp;
    dscale_tmp.Resize({NxC});
    dev_ctx.template Alloc<AccT>(&dscale_tmp);
    set_zero_AccT(dev_ctx, &dscale_tmp, static_cast<AccT>(0));
    AccT *dscale_tmp_data = dscale_tmp.data<AccT>();

    AccT *dscale_data = dev_ctx.template Alloc<AccT>(dscale);
    set_zero_AccT(dev_ctx, dscale, static_cast<AccT>(0));
    DoubleGradComputeDScale<T, AccT, block>
        <<<grid, block, 0, dev_ctx.stream()>>>(x_data,
                                               mean_data,
                                               variance_data,
                                               ddx_data,
                                               dy_data,
                                               C,
                                               sample_size,
                                               epsilon,
                                               dscale_tmp_data);
    add_param<AccT, block, false><<<grid1, block, 0, dev_ctx.stream()>>>(
        dscale_tmp.data<AccT>(), dscale->data<AccT>(), N, C);
  }
  if (ddy) {
    T *ddy_data = dev_ctx.template Alloc<T>(ddy);
    set_zero(dev_ctx, ddy, static_cast<T>(0));
    DoubleGradComputeDDY<T, AccT, block>
        <<<grid, block, 0, dev_ctx.stream()>>>(x_data,
                                               mean_data,
                                               variance_data,
                                               ddscale_data,
                                               ddbias_data,
                                               ddx_data,
                                               scale_data,
                                               C,
                                               sample_size,
                                               epsilon,
                                               ddy_data);
  }
}
}  // namespace phi

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(instance_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormGradKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(instance_norm_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormDoubleGradKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#else
PD_REGISTER_KERNEL(instance_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(instance_norm_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::InstanceNormDoubleGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#endif
