#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/layer_norm_kernel.h"
#include "gflags/gflags.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/layer_norm_impl.cu.h"
#include "paddle/phi/kernels/funcs/layer_norm_util.h"

DECLARE_bool(use_fast_math);

namespace phi {

#ifdef PADDLE_WITH_CUDA

template <typename U>
__device__ inline void LegacyOnline(U val, U *mean, U *square, U *count) {
  *mean += val;
  *square += val * val;
  *count += 1;
}

template <typename U>
__device__ inline void LegacyOnline(
    U old_mean, U old_square, U old_cnt, U *mean, U *square, U *cnt) {
  *mean += old_mean;
  *square += old_square;
  *cnt += old_cnt;
}

template <typename U>
__device__ inline void LegacyWarpAllReduce(U *mean, U *square, U *count) {
  constexpr int kWarpSize = 32;
#pragma unroll
  for (int mask = (kWarpSize >> 1); mask > 0; mask >>= 1) {
    U b_mean = __shfl_down_sync(0xffffffff, *mean, mask);
    U b_square = __shfl_down_sync(0xffffffff, *square, mask);
    U b_cnt = __shfl_down_sync(0xffffffff, *count, mask);
    LegacyOnline<U>(b_mean, b_square, b_cnt, mean, square, count);
  }

  *mean = __shfl_sync(0xffffffff, *mean, 0, kWarpSize);
  *square = __shfl_sync(0xffffffff, *square, 0, kWarpSize);
  *count = __shfl_sync(0xffffffff, *count, 0, kWarpSize);
}

template <typename U>
__device__ inline void WelfordOnline(U val, U *mean, U *square, U *count) {
  *count += 1;
  U delta1 = val - *mean;
  *mean += delta1 / (*count);
  U delta2 = val - *mean;
  *square += delta1 * delta2;
}

template <typename U>
__device__ inline void WelfordOnline(
    U b_mean, U b_square, U b_cnt, U *mean, U *square, U *count) {
  if (b_cnt == 0) {
    return;
  }

  U new_cnt = *count + b_cnt;
  U nb_n = b_cnt / new_cnt;
  U delta = b_mean - *mean;
  *mean += delta * nb_n;
  *square += b_square + delta * delta * (*count) * nb_n;
  *count = new_cnt;
}

template <typename U>
__device__ inline void WelfordWarpAllReduce(U *mean, U *square, U *count) {
  constexpr int kWarpSize = 32;
#pragma unroll
  for (int mask = 1; mask < kWarpSize; mask *= 2) {
    U b_mean = __shfl_down_sync(0xffffffff, *mean, mask);
    U b_square = __shfl_down_sync(0xffffffff, *square, mask);
    U b_cnt = __shfl_down_sync(0xffffffff, *count, mask);
    WelfordOnline<U>(b_mean, b_square, b_cnt, mean, square, count);
  }

  *mean = __shfl_sync(0xffffffff, *mean, 0, kWarpSize);
  *square = __shfl_sync(0xffffffff, *square, 0, kWarpSize);
  *count = __shfl_sync(0xffffffff, *count, 0, kWarpSize);
}

template <int VecSize>
struct ThreadAssigner {
  __device__ __forceinline__ int operator()(const int cols,
                                            const int cols_per_thread,
                                            int32_t *last_tid_idx) {
    return cols_per_thread;
  }
};

template <>
struct ThreadAssigner<1> {
  __device__ inline int operator()(const int cols,
                                   const int cols_per_thread,
                                   int *last_tid_idx) {
    int cols_this_thread = cols_per_thread;
    int last_tid = (cols / cols_per_thread);
    *last_tid_idx = last_tid;
    if (threadIdx.x == last_tid) {
      cols_this_thread = cols - cols_per_thread * last_tid;
    } else if (threadIdx.x > last_tid) {
      cols_this_thread = 0;
    }
    return cols_this_thread;
  }
};

template <typename T, typename U, int VecSize>
struct LayerNormDataReader {
  __device__ inline void operator()(const T *__restrict__ row_src,
                                    U *buffer,
                                    const int last_tid_idx,
                                    const int read_times,
                                    const int cols_this_thread) {
    using VecT = phi::AlignedVector<T, VecSize>;
    const VecT *__restrict__ v_src =
        reinterpret_cast<const VecT *__restrict__>(row_src);

    for (int i = 0; i < read_times; ++i) {
      VecT temp_src = v_src[threadIdx.x + i * blockDim.x];
#pragma unroll
      for (int j = 0; j < VecSize; ++j) {
        buffer[i * VecSize + j] = static_cast<U>(temp_src[j]);
      }
    }
  }
};

template <typename T, typename U>
struct LayerNormDataReader<T, U, 1> {
  __device__ inline void operator()(const T *__restrict__ row_src,
                                    U *buffer,
                                    const int last_tid_idx,
                                    const int read_times,
                                    const int cols_this_thread) {
    // read_time is just cols_per_thread while VecSize is 1.
    if (threadIdx.x < last_tid_idx) {
      for (int i = 0; i < cols_this_thread; ++i) {
        buffer[i] = static_cast<U>(row_src[threadIdx.x + last_tid_idx * i]);
      }
    } else {
      for (int i = 0; i < cols_this_thread; ++i) {
        buffer[i] = static_cast<U>(row_src[i + read_times * last_tid_idx]);
      }
    }
  }
};

template <typename T, typename U, bool IsSameType, int VecSize>
struct LayerNormDataWritter {
  __device__ inline void operator()(
      T *__restrict__ row_dst,
      const U *__restrict__ buffer,
      const funcs::LayerNormScaleBiasT<T, U, IsSameType> *__restrict__ scale,
      const funcs::LayerNormScaleBiasT<T, U, IsSameType> *__restrict__ bias,
      const U row_mean,
      const U row_inv_var,
      const int write_times,
      const int cols_this_thread,
      const int last_tid_idx,
      const bool valid_scale,
      const bool valid_bias) {
    using VecT = phi::AlignedVector<T, VecSize>;
    using ScaleT = funcs::LayerNormScaleBiasT<T, U, IsSameType>;
    using VecScaleT = phi::AlignedVector<ScaleT, VecSize>;
    VecT *v_dst = reinterpret_cast<VecT *>(row_dst);

    // cols_this_thread is just cols_per_thread
    if ((!valid_scale) && (!valid_bias)) {
      for (int i = 0; i < write_times; ++i) {
        VecT temp_dst;
#pragma unroll
        for (int j = 0; j < VecSize; ++j) {
          temp_dst[j] = static_cast<T>((buffer[i * VecSize + j] - row_mean) *
                                       row_inv_var);
        }
        v_dst[threadIdx.x + blockDim.x * i] = temp_dst;
      }
    } else {
      const VecScaleT *__restrict__ v_scale =
          reinterpret_cast<const VecScaleT *__restrict__>(scale);
      const VecScaleT *__restrict__ v_bias =
          reinterpret_cast<const VecScaleT *__restrict__>(bias);
      if (valid_scale && valid_bias) {
        for (int i = 0; i < write_times; ++i) {
          int idx = threadIdx.x + blockDim.x * i;
          VecT temp_dst;
          VecScaleT temp_v_scale = v_scale[idx];
          VecScaleT temp_v_bias = v_bias[idx];
#pragma unroll
          for (int j = 0; j < VecSize; ++j) {
            temp_dst[j] = static_cast<T>(
                static_cast<U>(temp_v_scale[j]) *
                    (buffer[i * VecSize + j] - row_mean) * row_inv_var +
                static_cast<U>(temp_v_bias[j]));
          }
          v_dst[idx] = temp_dst;
        }
      } else {
        if (valid_scale) {
          for (int i = 0; i < write_times; ++i) {
            int idx = threadIdx.x + blockDim.x * i;
            VecT temp_dst;
            VecScaleT temp_v_scale = v_scale[idx];
#pragma unroll
            for (int j = 0; j < VecSize; ++j) {
              temp_dst[j] = static_cast<T>(
                  static_cast<U>(temp_v_scale[j]) *
                  (buffer[i * VecSize + j] - row_mean) * row_inv_var);
            }
            v_dst[idx] = temp_dst;
          }
        } else {
          for (int i = 0; i < write_times; ++i) {
            int idx = threadIdx.x + blockDim.x * i;
            VecT temp_dst;
            VecScaleT temp_v_bias = v_bias[idx];
#pragma unroll
            for (int j = 0; j < VecSize; ++j) {
              temp_dst[j] = static_cast<T>(
                  (buffer[i * VecSize + j] - row_mean) * row_inv_var +
                  static_cast<U>(temp_v_bias[j]));
            }
            v_dst[idx] = temp_dst;
          }
        }
      }
    }
  }
};

template <typename T, typename U, bool IsSameType>
struct LayerNormDataWritter<T, U, IsSameType, 1> {
  __device__ __forceinline__ void operator()(
      T *__restrict__ row_dst,
      U *__restrict__ buffer,
      const funcs::LayerNormScaleBiasT<T, U, IsSameType> *__restrict__ scale,
      const funcs::LayerNormScaleBiasT<T, U, IsSameType> *__restrict__ bias,
      const U row_mean,
      const U row_inv_var,
      const int write_times,
      const int cols_this_thread,
      const int last_tid_idx,
      const bool valid_scale,
      const bool valid_bias) {
    // write_times is just col_per_thread.
    if ((!valid_scale) && (!valid_bias)) {
      if (threadIdx.x < last_tid_idx) {
        for (int i = 0; i < cols_this_thread; ++i) {
          row_dst[threadIdx.x + last_tid_idx * i] =
              (buffer[i] - row_mean) * row_inv_var;
        }
      } else {
        for (int i = 0; i < cols_this_thread; ++i) {
          row_dst[last_tid_idx * write_times + i] =
              (buffer[i] - row_mean) * row_inv_var;
        }
      }
    } else if (valid_scale && valid_bias) {
      if (threadIdx.x < last_tid_idx) {
        for (int i = 0; i < cols_this_thread; ++i) {
          int idx = threadIdx.x + last_tid_idx * i;
          row_dst[idx] =
              static_cast<T>(static_cast<U>(scale[idx]) *
                                 (buffer[i] - row_mean) * row_inv_var +
                             static_cast<U>(bias[idx]));
        }
      } else {
        for (int i = 0; i < cols_this_thread; ++i) {
          int idx = last_tid_idx * write_times + i;
          row_dst[idx] =
              static_cast<T>(static_cast<U>(scale[idx]) *
                                 (buffer[i] - row_mean) * row_inv_var +
                             static_cast<U>(bias[idx]));
        }
      }
    } else {
      if (valid_scale) {
        if (threadIdx.x < last_tid_idx) {
          for (int i = 0; i < cols_this_thread; ++i) {
            int idx = threadIdx.x + last_tid_idx * i;
            row_dst[idx] = static_cast<T>(static_cast<U>(scale[idx]) *
                                          (buffer[i] - row_mean) * row_inv_var);
          }
        } else {
          for (int i = 0; i < cols_this_thread; ++i) {
            int idx = last_tid_idx * write_times + i;
            row_dst[idx] = static_cast<T>(static_cast<U>(scale[idx]) *
                                          (buffer[i] - row_mean) * row_inv_var);
          }
        }
      } else {
        if (threadIdx.x < last_tid_idx) {
          for (int i = 0; i < cols_this_thread; ++i) {
            int idx = threadIdx.x + last_tid_idx * i;
            row_dst[idx] = static_cast<T>((buffer[i] - row_mean) * row_inv_var +
                                          static_cast<U>(bias[idx]));
          }
        } else {
          for (int i = 0; i < cols_this_thread; ++i) {
            int idx = last_tid_idx * write_times + i;
            row_dst[idx] = static_cast<T>((buffer[i] - row_mean) * row_inv_var +
                                          static_cast<U>(bias[idx]));
          }
        }
      }
    }
  }
};

template <typename IndexT,
          typename T,
          typename U,
          bool IsSameType,
          int VecSize,
          bool UseWelford = true>
__global__ void LayerNormFwdWithWelford(
    const T *__restrict__ src_data,
    T *dst_data,
    const funcs::LayerNormScaleBiasT<T, U, IsSameType> *__restrict__ scale,
    const funcs::LayerNormScaleBiasT<T, U, IsSameType> *__restrict__ bias,
    U *mean,
    U *var,
    const U epsilon,
    const IndexT rows,
    const int32_t cols,
    const int32_t cols_per_thread,
    const bool valid_scale,
    const bool valid_bias) {
  constexpr int kWarpSize = 32;
  int last_tid_idx = 0;  // For condition once vecSize is 1.
  IndexT row_offset = blockIdx.x * blockDim.y + threadIdx.y;
  int cols_this_thread =
      ThreadAssigner<VecSize>()(cols, cols_per_thread, &last_tid_idx);
  int read_times = cols_per_thread / VecSize;

  if (row_offset < rows) {
    U buffer[kWarpSize];
    U tid_cnt = static_cast<U>(0);
    U tid_mean = static_cast<U>(0);
    U tid_square = static_cast<U>(0);

    const T *__restrict__ row_src = src_data + row_offset * cols;
    T *row_dst = dst_data + row_offset * cols;
    LayerNormDataReader<T, U, VecSize>()(
        row_src, buffer, last_tid_idx, read_times, cols_this_thread);

    for (int i = 0; i < cols_this_thread; i++) {
      if constexpr (UseWelford) {
        WelfordOnline<U>(buffer[i], &tid_mean, &tid_square, &tid_cnt);
      } else {
        LegacyOnline<U>(buffer[i], &tid_mean, &tid_square, &tid_cnt);
      }
    }

    U warp_cnt = tid_cnt;
    U warp_mean = tid_mean;
    U warp_square = tid_square;

    if constexpr (UseWelford) {
      WelfordWarpAllReduce<U>(&warp_mean, &warp_square, &warp_cnt);
    } else {
      LegacyWarpAllReduce<U>(&warp_mean, &warp_square, &warp_cnt);
      warp_mean = warp_mean / warp_cnt;
    }

    U row_variance;
    U row_inv_var;

    if constexpr (UseWelford) {
      row_variance = max(warp_square / warp_cnt, 0.f);
      row_inv_var = funcs::rsqrt_(row_variance + epsilon);
    } else {
      row_variance = max(warp_square / warp_cnt - warp_mean * warp_mean, 0.f);
      row_inv_var = funcs::rsqrt_(row_variance + epsilon);
    }

    // TODO(limingshu): make code below vectorization.
    if (threadIdx.x == 0) {
      // warp_mean is just row_mean here.
      mean[row_offset] = warp_mean;
      var[row_offset] = row_variance;
    }
    LayerNormDataWritter<T, U, IsSameType, VecSize>()(row_dst,
                                                      buffer,
                                                      scale,
                                                      bias,
                                                      warp_mean,
                                                      row_inv_var,
                                                      read_times,
                                                      cols_this_thread,
                                                      last_tid_idx,
                                                      valid_scale,
                                                      valid_bias);
  }
}

template <typename Context, typename T, typename U>
void LaunchLayerNormKernel(const Context &dev_ctx,
                           const T *x_data,
                           T *y_data,
                           const void *void_scale_data,
                           const void *void_bias_data,
                           U *mean_data,
                           U *var_data,
                           float epsilon,
                           const int64_t rows,
                           const int cols,
                           const bool valid_scale,
                           const bool valid_bias,
                           const bool is_same_type) {
  constexpr int WarpSize = 32;
  constexpr int RowPerBlock = 4;
  int64_t block_size = (rows + (RowPerBlock - 1)) / RowPerBlock;
  dim3 threads(WarpSize, RowPerBlock, 1);

  int vec_size = 1;
  int cols_per_thread = (cols + (WarpSize - 1)) / WarpSize;
  if (cols_per_thread > 1 && (cols % WarpSize == 0)) {
    int data_vec_size = 0;
    uint64_t addr = (reinterpret_cast<uint64_t>(x_data) |
                     reinterpret_cast<uint64_t>(y_data));
    if (valid_bias || valid_scale) {
      if (is_same_type) {
        addr = valid_scale
                   ? (addr | reinterpret_cast<uint64_t>(void_scale_data))
                   : addr;
        addr = valid_bias ? (addr | reinterpret_cast<uint64_t>(void_bias_data))
                          : addr;
        data_vec_size = phi::GetVectorizedSize<T>(reinterpret_cast<T *>(addr));
      } else {
        uint64_t bias_addr = reinterpret_cast<uint64_t>(void_bias_data);
        uint64_t attr_addr = valid_scale
                                 ? reinterpret_cast<uint64_t>(void_scale_data)
                                 : bias_addr;
        attr_addr = valid_bias
                        ? (valid_scale ? (attr_addr | bias_addr) : attr_addr)
                        : attr_addr;
        data_vec_size = std::min(
            phi::GetVectorizedSize<T>(reinterpret_cast<T *>(addr)),
            phi::GetVectorizedSize<U>(reinterpret_cast<U *>(attr_addr)));
      }
    }
    for (int size = data_vec_size; size > 0; size /= 2) {
      if (cols_per_thread % size == 0) {
        vec_size = size;
        break;
      }
    }
  }

#define IMPL_LAYER_NORM_WELFORD_CASE(index_t, scale_t, is_same_, vec_size_) \
  case (vec_size_): {                                                       \
    LayerNormFwdWithWelford<index_t, T, U, is_same_, vec_size_>             \
        <<<block_size, threads, 0, dev_ctx.stream()>>>(                     \
            x_data,                                                         \
            y_data,                                                         \
            static_cast<const scale_t *>(void_scale_data),                  \
            static_cast<const scale_t *>(void_bias_data),                   \
            mean_data,                                                      \
            var_data,                                                       \
            static_cast<const U>(epsilon),                                  \
            rows,                                                           \
            cols,                                                           \
            cols_per_thread,                                                \
            valid_scale,                                                    \
            valid_bias);                                                    \
  } break

#define IMPL_LAYER_NORM_WELFORD(index_t, scale_t, is_same_)    \
  IMPL_LAYER_NORM_WELFORD_CASE(index_t, scale_t, is_same_, 4); \
  IMPL_LAYER_NORM_WELFORD_CASE(index_t, scale_t, is_same_, 2); \
  IMPL_LAYER_NORM_WELFORD_CASE(index_t, scale_t, is_same_, 1);

  if (rows < std::numeric_limits<int32_t>::max()) {
    if (is_same_type) {
      switch (vec_size) { IMPL_LAYER_NORM_WELFORD(int32_t, T, true); }
    } else {
      switch (vec_size) { IMPL_LAYER_NORM_WELFORD(int32_t, U, false); }
    }
  } else {
    if (is_same_type) {
      switch (vec_size) { IMPL_LAYER_NORM_WELFORD(int64_t, T, true); }
    } else {
      switch (vec_size) { IMPL_LAYER_NORM_WELFORD(int64_t, U, false); }
    }
  }
#undef IMPL_LAYER_NORM_WELFORD_CASE
#undef IMPL_LAYER_NORM_WELFORD
}
#endif  // PADDLE_WITH_CUDA

template <typename T, typename U>
void LayerNormDirectCUDAFunctor<T, U>::operator()(gpuStream_t stream,
                                                  const T *input,
                                                  std::vector<int> input_shape,
                                                  const U *bias,
                                                  const U *scale,
                                                  T *output,
                                                  U *mean,
                                                  U *variance,
                                                  int begin_norm_axis,
                                                  float eps) {
  const auto x_dims = phi::make_ddim(input_shape);
  auto matrix_dim = phi::flatten_to_2d(x_dims, begin_norm_axis);
  int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
  int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);
  switch (phi::funcs::GetDesiredBlockDim(feature_size)) {
    FIXED_BLOCK_DIM_CASE(
        phi::funcs::LayerNormForward<T, U, kBlockDim>
        <<<batch_size, kBlockDim, 0, stream>>>(
            input, scale, bias, output, mean, variance, eps, feature_size));
    default:
      PADDLE_THROW(phi::errors::InvalidArgument(
          "Product from begin_norm_axis to end in layer_norm must be larger "
          "than 1"));
      break;
  }
}

template class LayerNormDirectCUDAFunctor<float, float>;
template class LayerNormDirectCUDAFunctor<double, double>;
#if defined(PADDLE_WITH_CUDA) && !defined(PADDLE_WITH_HIP)
template class LayerNormDirectCUDAFunctor<half, float>;
#endif

template <typename T, typename Context>
void LayerNormKernel(const Context &dev_ctx,
                     const DenseTensor &x,
                     const paddle::optional<DenseTensor> &scale_opt,
                     const paddle::optional<DenseTensor> &bias_opt,
                     float epsilon,
                     int begin_norm_axis,
                     DenseTensor *y,
                     DenseTensor *mean,
                     DenseTensor *var) {
  using U = phi::funcs::LayerNormParamType<T>;
  auto *scale = scale_opt.get_ptr();
  auto *bias = bias_opt.get_ptr();

  const auto x_dims = x.dims();
  auto *x_data = x.data<T>();
  auto *y_data = dev_ctx.template Alloc<T>(y);
  auto *mean_data = dev_ctx.template Alloc<U>(mean);
  auto *var_data = dev_ctx.template Alloc<U>(var);

  bool valid_scale = (scale != nullptr);
  bool valid_bias = (bias != nullptr);
  auto *void_scale_data = valid_scale ? scale->data() : nullptr;
  auto *void_bias_data = valid_bias ? bias->data() : nullptr;

  auto x_dtype = x.dtype();
  phi::DataType scale_bias_dtype;
  if (valid_scale) {
    scale_bias_dtype = scale->dtype();
    if (valid_bias) {
      PADDLE_ENFORCE_EQ(
          scale->dtype(),
          bias->dtype(),
          phi::errors::InvalidArgument("This Scale and Bias of layer_norm op "
                                       "should have the same data type."));
    }
  } else {
    scale_bias_dtype = valid_bias ? bias->dtype() : x_dtype;
  }

  bool is_scale_bias_same_dtype_with_x = x_dtype == scale_bias_dtype;
  if (!is_scale_bias_same_dtype_with_x) {
    PADDLE_ENFORCE_EQ(scale_bias_dtype,
                      phi::CppTypeToDataType<U>::Type(),
                      phi::errors::InvalidArgument(
                          "Unsupported data type of Scale and Bias"));
  }

  auto matrix_dim = phi::flatten_to_2d(x_dims, begin_norm_axis);
  int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
  int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);
  auto stream = dev_ctx.stream();

#define PADDLE_LAUNCH_LAYERNORM_FWD(ScaleBiasT, IsScaleBiasSameDTypeWithX) \
  do {                                                                     \
    switch (phi::funcs::GetDesiredBlockDim(feature_size)) {                \
      FIXED_BLOCK_DIM_CASE(                                                \
          phi::funcs::                                                     \
              LayerNormForward<T, U, kBlockDim, IsScaleBiasSameDTypeWithX> \
          <<<batch_size, kBlockDim, 0, stream>>>(                          \
              x_data,                                                      \
              static_cast<const ScaleBiasT *>(void_scale_data),            \
              static_cast<const ScaleBiasT *>(void_bias_data),             \
              y_data,                                                      \
              mean_data,                                                   \
              var_data,                                                    \
              epsilon,                                                     \
              feature_size));                                              \
      default:                                                             \
        PADDLE_THROW(phi::errors::InvalidArgument(                         \
            "Product from begin_norm_axis to end must be larger than 1")); \
        break;                                                             \
    }                                                                      \
  } while (0)

#define PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, feature_size)          \
  case (feature_size): {                                                     \
    constexpr int WARPS_N = feature_size < 1024 ? 1 : (feature_size / 1024); \
    constexpr int WARPS_M = 4 / WARPS_N;                                     \
    const int THREADS_PER_WARP = 32;                                         \
    const int BYTES_PER_LDG = 16;                                            \
    const int VecSize = BYTES_PER_LDG / sizeof(T);                           \
    const int THREADS_PER_CTA = WARPS_N * THREADS_PER_WARP * WARPS_M;        \
    const int ROWS_PER_CTA = WARPS_M;                                        \
    const int grid = static_cast<int>(                                       \
        std::ceil(batch_size / static_cast<float>(ROWS_PER_CTA)));           \
    phi::funcs::fast_ln_fwd_kernel<T,                                        \
                                   U,                                        \
                                   ScaleT,                                   \
                                   VecSize,                                  \
                                   WARPS_M,                                  \
                                   WARPS_N,                                  \
                                   BYTES_PER_LDG>                            \
        <<<grid, THREADS_PER_CTA, 0, stream>>>(                              \
            batch_size,                                                      \
            feature_size,                                                    \
            epsilon,                                                         \
            x_data,                                                          \
            static_cast<const ScaleT *>(void_scale_data),                    \
            static_cast<const ScaleT *>(void_bias_data),                     \
            mean_data,                                                       \
            var_data,                                                        \
            y_data);                                                         \
  } break

#define PADDLE_LAUNCH_FAST_LAYERNORM_FWD(ScaleT)       \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 768);  \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 1024); \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 1280); \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 1536); \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 1792); \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 2048); \
  PADDLE_LAUNCH_FAST_LAYERNORM_FWD_BASE(ScaleT, 4096)

#ifdef PADDLE_WITH_CUDA
  bool can_call_fast_kernel = false;
  if ((feature_size >= 768 && feature_size <= 2048 && feature_size % 256 == 0 ||
       feature_size == 4096) &&
      scale != nullptr && bias != nullptr) {
    // can_call_fast_kernel = true;
    can_call_fast_kernel = false;
  }

  if (can_call_fast_kernel) {
    if (is_scale_bias_same_dtype_with_x) {
      switch (feature_size) {
        PADDLE_LAUNCH_FAST_LAYERNORM_FWD(T);
        default:
          PADDLE_THROW(phi::errors::InvalidArgument(
              "Only when feature_size is from 256 to 4096 and is diviaible by "
              "256 is supported "
              "now"));
          break;
      }
    } else {
      switch (feature_size) {
        PADDLE_LAUNCH_FAST_LAYERNORM_FWD(U);
        default:
          PADDLE_THROW(phi::errors::InvalidArgument(
              "Only when feature_size is from 256 to 4096 and is diviaible by "
              "is supported "
              "now"));
          break;
      }
    }
  } else {
    // WarpShuffle intrinsics is involved in LaunchLayerNormKernel.
    if (FLAGS_use_fast_math && feature_size <= 1024 &&
        (!std::is_same<T, int8_t>::value)) {
      LaunchLayerNormKernel<Context, T, U>(dev_ctx,
                                           x_data,
                                           y_data,
                                           void_scale_data,
                                           void_bias_data,
                                           mean_data,
                                           var_data,
                                           epsilon,
                                           batch_size,
                                           feature_size,
                                           valid_scale,
                                           valid_bias,
                                           is_scale_bias_same_dtype_with_x);
    } else {
#endif
      if (is_scale_bias_same_dtype_with_x) {
        PADDLE_LAUNCH_LAYERNORM_FWD(T, true);
      } else {
        PADDLE_LAUNCH_LAYERNORM_FWD(U, false);
      }
#ifdef PADDLE_WITH_CUDA
    }
  }
#endif

#undef PADDLE_LAUNCH_LAYERNORM_FWD
#undef PADDLE_LAUNCH_FAST_LAYERNORM_FWD
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(layer_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::LayerNormKernel,
                   float,
                   phi::dtype::float16) {
  kernel->OutputAt(1).SetDataType(phi::DataType::UNDEFINED);
  kernel->OutputAt(2).SetDataType(phi::DataType::UNDEFINED);
}
#elif CUDNN_VERSION_MIN(8, 1, 0)
PD_REGISTER_KERNEL(layer_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::LayerNormKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->OutputAt(1).SetDataType(phi::DataType::UNDEFINED);
  kernel->OutputAt(2).SetDataType(phi::DataType::UNDEFINED);
}
#else
PD_REGISTER_KERNEL(layer_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::LayerNormKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->OutputAt(1).SetDataType(phi::DataType::UNDEFINED);
  kernel->OutputAt(2).SetDataType(phi::DataType::UNDEFINED);
}
#endif
