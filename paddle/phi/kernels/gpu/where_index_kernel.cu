// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/select_impl.cu.h"
#include "paddle/phi/kernels/where_index_kernel.h"

namespace phi {
template <typename MaskT, typename IndexT, typename OutT>
struct IndexFunctor {
  IndexT strides[phi::DDim::kMaxRank];
  int rank;

  explicit IndexFunctor(const phi::DDim &in_dims) {
    rank = in_dims.size();
    // Get strides according to in_dims
    strides[0] = 1;
    for (IndexT i = 1; i < rank; i++) {
      strides[i] = strides[i - 1] * in_dims[rank - i];
    }
  }

  HOSTDEVICE inline void operator()(OutT *out,
                                    const MaskT *mask,
                                    const IndexT *index,
                                    const int num) {
    int store_fix = 0;
    for (int idx = 0; idx < num; idx++) {
      if (mask[idx]) {
        IndexT data_index = index[idx];
        // get index
        for (int rank_id = rank - 1; rank_id >= 0; --rank_id) {
          out[store_fix] = static_cast<OutT>(data_index / strides[rank_id]);
          data_index = data_index % strides[rank_id];
          store_fix++;
        }
      }
    }
  }
};

template <typename T, typename Context>
void WhereIndexKernel(const Context &dev_ctx,
                      const DenseTensor &condition,
                      DenseTensor *out) {
  DenseTensor in_data;
  auto dims = condition.dims();
  using Functor = IndexFunctor<T, int64_t, int64_t>;
  Functor index_functor = Functor(dims);
  phi::funcs::SelectKernel<T, T, int64_t, 0, Functor>(
      dev_ctx, condition, in_data, out, index_functor);
}
}  // namespace phi

PD_REGISTER_KERNEL(where_index,
                   GPU,
                   ALL_LAYOUT,
                   phi::WhereIndexKernel,
                   int64_t,
                   int,
                   int16_t,
                   bool,
                   float,
                   double) {}
