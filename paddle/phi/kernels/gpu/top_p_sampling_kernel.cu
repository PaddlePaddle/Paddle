#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/top_p_sampling_kernel.h"

#ifdef PADDLE_WITH_HIP
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#else
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <hipcub/hipcub.hpp>
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_device_function.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/gather.cu.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/top_k_function_cuda.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

#ifdef PADDLE_WITH_HIP
#define GPU(str) hip##str
#else
#define GPU(str) cu##str
#endif

// #define DEBUG_TOPP

namespace phi {

template <typename T>
struct DataTypeTraits {
  using DataType = T;
};

template <>
struct DataTypeTraits<phi::dtype::float16> {
  using DataType = half;
};

#ifdef PADDLE_CUDA_BF16
template <>
struct DataTypeTraits<phi::dtype::bfloat16> {
  using DataType = __hip_bfloat16;
};
#endif

#define FINAL_MASK 0xFFFFFFFF

#define FIXED_BLOCK_DIM_BASE(dim, ...) \
  case (dim): {                        \
    constexpr auto kBlockDim = (dim);  \
    __VA_ARGS__;                       \
  } break

#ifdef PADDLE_WITH_HIP
#define WARP_SIZE 64
#define FIXED_BLOCK_DIM(...)                 \
  FIXED_BLOCK_DIM_BASE(1024, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(512, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);
#else
#define WARP_SIZE 32
#define FIXED_BLOCK_DIM(...)                 \
  FIXED_BLOCK_DIM_BASE(1024, ##__VA_ARGS__); \
  FIXED_BLOCK_DIM_BASE(512, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(256, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(128, ##__VA_ARGS__);  \
  FIXED_BLOCK_DIM_BASE(64, ##__VA_ARGS__);   \
  FIXED_BLOCK_DIM_BASE(32, ##__VA_ARGS__)
#endif

struct SegmentOffsetIter {
  explicit SegmentOffsetIter(int num_cols) : num_cols_(num_cols) {}

  __host__ __device__ __forceinline__ int operator()(int idx) const {
    return idx * num_cols_;
  }

  int num_cols_;
};

template <typename T>
struct Pair {
  __device__ __forceinline__ Pair() {}
  __device__ __forceinline__ Pair(T value, int id) : v(value), id(id) {}

  __device__ __forceinline__ void set(T value, int id) {
    this->v = value;
    this->id = id;
  }

  __device__ __forceinline__ void operator=(const Pair<T>& in) {
    v = in.v;
    id = in.id;
  }

  __device__ __forceinline__ bool operator<(const T value) const {
    return (static_cast<float>(v) < static_cast<float>(value));
  }

  __device__ __forceinline__ bool operator>(const T value) const {
    return (static_cast<float>(v) > static_cast<float>(value));
  }
  __device__ __forceinline__ bool operator<(const Pair<T>& in) const {
    return (static_cast<float>(v) < static_cast<float>(in.v)) ||
           ((static_cast<float>(v) == static_cast<float>(in.v)) &&
            (id > in.id));
  }

  __device__ __forceinline__ bool operator>(const Pair<T>& in) const {
    return (static_cast<float>(v) > static_cast<float>(in.v)) ||
           ((static_cast<float>(v) == static_cast<float>(in.v)) &&
            (id < in.id));
  }

  T v;
  int id;
};

int GetBlockSize(int vocab_size) {
  if (vocab_size > 512) {
    return 1024;
  } else if (vocab_size > 256) {
    return 512;
  } else if (vocab_size > 128) {
    return 256;
  } else if (vocab_size > 64) {
    return 128;
  } else {
    return 64;
  }
}

inline int div_up(int a, int n) { return (a + n - 1) / n; }

template <typename T>
__device__ __forceinline__ void AddTo(Pair<T> topk[],
                                      const Pair<T>& p,
                                      int beam_size) {
  for (int k = beam_size - 2; k >= 0; k--) {
    if (topk[k] < p) {
      topk[k + 1] = topk[k];
    } else {
      topk[k + 1] = p;
      return;
    }
  }
  topk[0] = p;
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(
    Pair<T> topk[], const T* src, int idx, int dim, int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      AddTo<T>(topk, tmp, beam_size);
    }
    idx += BlockSize;
  }
}

template <typename T, int BlockSize>
__device__ __forceinline__ void GetTopK(Pair<T> topk[],
                                        const T* src,
                                        int idx,
                                        int dim,
                                        const Pair<T>& max,
                                        int beam_size) {
  while (idx < dim) {
    if (topk[beam_size - 1] < src[idx]) {
      Pair<T> tmp(src[idx], idx);
      if (tmp < max) {
        AddTo<T>(topk, tmp, beam_size);
      }
    }
    idx += BlockSize;
  }
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void ThreadGetTopK(Pair<T> topk[],
                                              int* beam,
                                              int beam_size,
                                              const T* src,
                                              bool* firstStep,
                                              bool* is_empty,
                                              Pair<T>* max,
                                              int dim,
                                              const int tid) {
  if (*beam > 0) {
    int length = (*beam) < beam_size ? *beam : beam_size;
    if (*firstStep) {
      *firstStep = false;
      GetTopK<T, BlockSize>(topk, src, tid, dim, length);
    } else {
      for (int k = 0; k < MaxLength; k++) {
        if (k < MaxLength - (*beam)) {
          topk[k] = topk[k + *beam];
        } else {
          topk[k].set(std::numeric_limits<T>::min(), -1);
        }
      }
      if (!(*is_empty)) {
        GetTopK<T, BlockSize>(
            topk + MaxLength - *beam, src, tid, dim, *max, length);
      }
    }

    *max = topk[MaxLength - 1];
    if ((*max).id == -1) *is_empty = true;
    *beam = 0;
  }
}

template <typename T>
__forceinline__ __device__ Pair<T> WarpReduce(Pair<T> input) {
#pragma unroll
  for (int offset = 16; offset > 0; offset >>= 1) {
    T tmp_val =
        phi::backends::gpu::CudaShuffleDownSync(FINAL_MASK, input.v, offset);
    int tmp_id =
        phi::backends::gpu::CudaShuffleDownSync(FINAL_MASK, input.id, offset);
    if (static_cast<float>(input.v) < static_cast<float>(tmp_val)) {
      input.v = tmp_val;
      input.id = tmp_id;
    }
  }
  return input;
}

template <typename T, int MaxLength, int BlockSize>
__device__ __forceinline__ void BlockReduce(Pair<T> shared_max[],
                                            Pair<T> topk[],
                                            Pair<T> beam_max[],
                                            int* beam,
                                            int* k,
                                            int* count,
                                            const int tid,
                                            const int wid,
                                            const int lane) {
  while (true) {
    __syncthreads();
    Pair<T> input_now = topk[0];
    input_now = WarpReduce(input_now);

    if (lane == 0) {
      shared_max[wid] = input_now;
    }
    __syncthreads();
    input_now = (tid < BlockSize / 32)
                    ? shared_max[lane]
                    : Pair<T>(std::numeric_limits<T>::min(), -1);
    if (wid == 0) {
      input_now = WarpReduce(input_now);
      if (lane == 0) shared_max[0] = input_now;
    }
    __syncthreads();
    if (tid == 0) {
      beam_max[*count] = shared_max[0];
      (*count)++;
    }
    int tid_max = shared_max[0].id % BlockSize;
    if (tid == tid_max) {
      (*beam)++;
    }
    if (--(*k) == 0) break;
    __syncthreads();

    if (tid == tid_max) {
      if (*beam < MaxLength) {
        topk[0] = topk[*beam];
      }
    }

    if (MaxLength < 5) {
      if (*beam >= MaxLength) break;
    } else {
#ifdef PADDLE_WITH_HIP
      unsigned mask = 0u;
      mask = __ballot(true);
      if (tid_max / WARP_SIZE == wid) {
        if (__shfl_down(*beam, tid_max % WARP_SIZE, WARP_SIZE) == MaxLength)
          break;
      }
#else
      unsigned mask = 0u;
      mask = __ballot_sync(FINAL_MASK, true);
      if (tid_max / WARP_SIZE == wid) {
        if (__shfl_down_sync(
                FINAL_MASK, *beam, tid_max % WARP_SIZE, WARP_SIZE) == MaxLength)
          break;
      }
#endif
    }
  }
}

template <typename T>
__device__ inline T exponential_transform(T val, T lambda) {
#if defined(__NVCC__) || defined(__HIPCC__)
  T log = -std::numeric_limits<T>::epsilon() / 2;
  if (val < static_cast<T>(1.) - std::numeric_limits<T>::epsilon() / 2) {
    if (std::is_same<T, double>::value) {
      log = logf(val);
    } else {
      log = __logf(val);
    }
  }
  return static_cast<T>(-1.0) / lambda * log;
#else
  return static_cast<T>(-1.0) / lambda * std::log(static_cast<T>(1.0) - val);
#endif
}

template <typename T, int MaxLength, int TopPBeamTopK, int BlockSize>
__global__ void KeMatrixTopPBeamTopK(const T* src,
                                     const T* threshold,
                                     GPU(randState_t) * states,
                                     T* top_ps,
                                     int64_t* out_id,  // topk id
                                     T* out_val,       // topk val
                                     int64_t* topk_ids,
                                     T* topk_scores,
                                     int vocab_size,
                                     int* count_iter,
                                     int* count_iter_begin,
                                     const int k,
                                     const bool need_batch_random) {
  const int tid = threadIdx.x;
  const int wid = tid / 32;
  const int lane = tid % 32;
  const int bid = blockIdx.x;
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;

  int top_num = TopPBeamTopK;
  float top_p_num = static_cast<float>(top_ps[bid]);
  const int offset = bid * vocab_size;
  int64_t* topk_ids_now = nullptr;
  T* topk_scores_now = nullptr;
  if (k > 0) {
    topk_ids_now = topk_ids + bid * k;
    topk_scores_now = topk_scores + bid * k;
  }

  __shared__ Pair<T> shared_max[BlockSize / 32];
  __shared__ Pair<T> beam_max[TopPBeamTopK];

  Pair<T> topk[MaxLength];
  int beam = MaxLength;
  Pair<T> max;
  bool is_empty = false;
  bool firststep = true;
  __shared__ int count;

  if (tid == 0) {
    count = 0;
  }

  for (int j = 0; j < MaxLength; j++) {
    topk[j].set(std::numeric_limits<T>::min(), -1);
  }

  while (top_num) {
    ThreadGetTopK<T, MaxLength, BlockSize>(topk,
                                           &beam,
                                           TopPBeamTopK,
                                           src + offset,
                                           &firststep,
                                           &is_empty,
                                           &max,
                                           vocab_size,
                                           tid);
    BlockReduce<T, MaxLength, BlockSize>(
        shared_max, topk, beam_max, &beam, &top_num, &count, tid, wid, lane);
  }
  if (tid == 0) {
    count_iter_begin[bid] = count_iter[bid];
    float top_p = top_ps[bid];
    float sum_prob = 0.0f;
    bool flag = false;
    float max_val = 0.f;
    int max_id = -1;
    for (int i = 0; i < TopPBeamTopK; i++) {
      if (i < k) {
        topk_ids_now[i] = static_cast<int64_t>(beam_max[i].id);
        topk_scores_now[i] = beam_max[i].v;
      }
      if (!flag) {
        float val = static_cast<float>(beam_max[i].v);
        sum_prob += val;
#ifdef PADDLE_WITH_HIP
        float random_ratio =
            exponential_transform(hiprand_uniform(states + bid), 1.0f);
#else
        float random_ratio =
            exponential_transform(hiprand_uniform(states + bid), 1.0f);
#endif
        float random_val = (val >= threshold_now ? val : 0.f) / random_ratio;
        if (max_val < random_val) {
          max_val = random_val;
          max_id = i;
        }
        if (sum_prob >= top_p) {
          flag = true;
          count_iter_begin[bid] += 1;
          if (max_id == -1) {
            // don't sample low score token
            out_id[bid] = static_cast<int64_t>(beam_max[0].id);
            out_val[bid] = beam_max[0].v;
          } else {
            out_id[bid] = static_cast<int64_t>(beam_max[max_id].id);
            out_val[bid] = beam_max[max_id].v;
          }
        }
      }
      if (flag && i >= k - 1) {
        break;
      }
    }
  }
}

template <typename T, int MaxLength, int TopPBeamTopK, int BlockSize>
__global__ void KeMatrixTopPBeamTopKFt(const T* src,
                                       const T* threshold,
                                       GPU(randState_t) * states,
                                       T* top_ps,
                                       int64_t* out_id,  // topk id
                                       T* out_val,       // topk val
                                       int64_t* topk_ids,
                                       T* topk_scores,
                                       int vocab_size,
                                       int* count_iter,
                                       int* count_iter_begin,
                                       const int k,
                                       const bool need_batch_random) {
  const int tid = threadIdx.x;
  const int wid = tid / 32;
  const int lane = tid % 32;
  const int bid = blockIdx.x;
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;

  int top_num = TopPBeamTopK;
  float top_p_num = static_cast<float>(top_ps[bid]);
  int64_t* topk_ids_now = nullptr;
  T* topk_scores_now = nullptr;
  if (k > 0) {
    topk_ids_now = topk_ids + bid * k;
    topk_scores_now = topk_scores + bid * k;
  }

  __shared__ Pair<T> shared_max[BlockSize / 32];
  __shared__ Pair<T> beam_max[TopPBeamTopK];

  Pair<T> topk[MaxLength];
  int beam = MaxLength;
  Pair<T> max;
  bool is_empty = false;
  bool firststep = true;
  __shared__ int count;

  if (tid == 0) {
    count = 0;
  }

  for (int j = 0; j < MaxLength; j++) {
    topk[j].set(std::numeric_limits<T>::min(), -1);
  }

  while (top_num) {
    ThreadGetTopK<T, MaxLength, BlockSize>(topk,
                                           &beam,
                                           TopPBeamTopK,
                                           src + bid * vocab_size,
                                           &firststep,
                                           &is_empty,
                                           &max,
                                           vocab_size,
                                           tid);
    BlockReduce<T, MaxLength, BlockSize>(
        shared_max, topk, beam_max, &beam, &top_num, &count, tid, wid, lane);
  }
  if (tid == 0) {
    count_iter_begin[bid] = count_iter[bid];
#ifdef PADDLE_WITH_HIP
    float rand_top_p = hiprand_uniform(states + bid) * top_p_num;
#else
    float rand_top_p = hiprand_uniform(states + bid) * top_p_num;
#endif
    top_ps[bid] = (T)rand_top_p;
    float sum_prob = 0.0f;
    bool flag = false;
    for (int i = 0; i < TopPBeamTopK; i++) {
      if (i < k) {
        topk_ids_now[i] = static_cast<int64_t>(beam_max[i].id);
        topk_scores_now[i] = beam_max[i].v;
      }
      if (!flag) {
        float val = static_cast<float>(beam_max[i].v);
        sum_prob += val;
#ifdef DEBUG_TOPP
        printf("bi: %d, top_p: %f, rand_top_p: %f, sum_prob: %f\n",
               bid,
               top_p_num,
               rand_top_p,
               sum_prob);
#endif
        if (sum_prob >= rand_top_p) {
          flag = true;
          count_iter_begin[bid] += 1;
          if (val < threshold_now) {
            // don't sample low score token
            int start_id = i == 0 ? 0 : i - 1;
            for (int j = start_id; j >= 0; j--) {
              float val_now = static_cast<float>(beam_max[j].v);
              if (val_now >= threshold_now || j == 0) {
                out_id[bid] = static_cast<int64_t>(beam_max[j].id);
                out_val[bid] = beam_max[j].v;
                break;
              }
            }
          } else {
            out_id[bid] = static_cast<int64_t>(beam_max[i].id);
            out_val[bid] = beam_max[i].v;
          }
        }
      }
      if (flag && i >= k - 1) {
        break;
      }
    }
  }
}

__global__ void SetCountIter(int* count_iter, int num) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + tid;
  for (int i = idx; i < num; i += gridDim.x * blockDim.x) {
    count_iter[i] = i;
  }
}

template <typename T>
__global__ void FillIndex(T* indices, T num_rows, T num_cols) {
  int col_id = threadIdx.x;
  int row_id = blockIdx.x;

  for (T j = row_id; j < num_rows; j += gridDim.x) {
    for (T i = col_id; i < num_cols; i += blockDim.x) {
      indices[j * num_cols + i] = i;
    }
  }
}

template <typename T, typename Context, int TopKMaxLength, int TopPBeamTopK>
void DispatchKeMatrixTopPBeamTopK(const Context& dev_ctx,
                                  const T* src,
                                  const T* threshold,
                                  GPU(randState_t) * states,
                                  T* top_ps,
                                  int64_t* out_id,  // topk id
                                  T* out_val,       // topk val
                                  int64_t* topk_ids,
                                  T* topk_scores,
                                  int vocab_size,
                                  int* count_iter,
                                  int* count_iter_begin,
                                  const int k,
                                  const int bs,
                                  const bool need_batch_random,
                                  const std::string& mode) {
  int BlockSize = GetBlockSize(vocab_size);
  if (mode == "truncate") {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          KeMatrixTopPBeamTopKFt<T, TopKMaxLength, TopPBeamTopK, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(src,
                                                   threshold,
                                                   states,
                                                   top_ps,
                                                   out_id,
                                                   out_val,
                                                   topk_ids,
                                                   topk_scores,
                                                   vocab_size,
                                                   count_iter,
                                                   count_iter_begin,
                                                   k,
                                                   need_batch_random));
      default:
        PD_THROW(
            "the input data shape has error in the topp_beam_topk kernel.");
    }
  } else {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          KeMatrixTopPBeamTopK<T, TopKMaxLength, TopPBeamTopK, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(src,
                                                   threshold,
                                                   states,
                                                   top_ps,
                                                   out_id,
                                                   out_val,
                                                   topk_ids,
                                                   topk_scores,
                                                   vocab_size,
                                                   count_iter,
                                                   count_iter_begin,
                                                   k,
                                                   need_batch_random));
      default:
        PD_THROW(
            "the input data shape has error in the topp_beam_topk kernel.");
    }
  }
}

struct BlockPrefixCallbackOp {
  // Running prefix
  float running_total;
  // Constructor
  __device__ BlockPrefixCallbackOp(float running_total)
      : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide
  // scan.
  __device__ float operator()(float block_aggregate) {
    float old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T>
__device__ T max_func(const T a, const T b) {
  return a > b ? a : b;
}

template <typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const {
    return max_func(a, b);
  }
};

template <typename T, int BLOCK_SIZE>
__global__ void topp_sampling(T* sorted_probs,
                              int64_t* sorted_id,
                              T* out_val,
                              int64_t* out_id,
                              const T* top_ps,
                              const T* threshold,
                              GPU(randState_t) * states,
                              const int p_num,
                              const int vocab_size,
                              const bool need_batch_random,
                              int* count_iter,
                              int* count_iter_begin) {
  __shared__ int stop_shared;
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  constexpr int NUM_WARPS = BLOCK_SIZE / 32;
  const int lane_id = tid % 32;
  const int warp_id = tid / 32;
  const float p_t = static_cast<float>(top_ps[bid]);
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;
  if (tid == 0) {
    stop_shared = 0;
  }
  if (count_iter_begin[bid] == count_iter[bid + 1]) {
    // topk
    return;
  }

  typedef hipcub::BlockScan<float, BLOCK_SIZE> BlockScan;
  typedef hipcub::BlockReduce<Pair<T>, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ typename BlockReduce::TempStorage temp_storage_reduce;

  // Initialize running total
  BlockPrefixCallbackOp prefix_op(0);

  int offset = bid * vocab_size;
#ifdef DEBUG_TOPP
  if (tid == 0) {
    printf(
        "first_elem1_1: %f, first_elem1_2: %f, first_id1_1: %d, first_id1_2: "
        "%d\n",
        static_cast<float>(sorted_probs[offset]),
        static_cast<float>(sorted_probs[offset + 1]),
        static_cast<int>(sorted_id[offset]),
        static_cast<int>(sorted_id[offset + 1]));
  }
#endif
  int end = ((vocab_size + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
  int i_activate = 0;
  float thread_offset = 0;
  Pair<T> max_thread_pair(static_cast<T>(0.), -1);
  for (int i = tid; i < end; i += BLOCK_SIZE) {
    float thread_count =
        (i < vocab_size) ? static_cast<float>(sorted_probs[offset + i]) : 0.f;
    BlockScan(temp_storage)
        .InclusiveSum(thread_count, thread_offset, prefix_op);

    if (thread_offset < p_t ||
        (thread_offset >= p_t && thread_offset - thread_count < p_t)) {
#ifdef PADDLE_WITH_HIP
      float random_ratio =
          exponential_transform(hiprand_uniform(states + bid), 1.0f);
#else
      float random_ratio =
          exponential_transform(hiprand_uniform(states + bid), 1.0f);
#endif
      float tmp_val =
          (thread_count >= threshold_now ? thread_count : 0.f) / random_ratio;
      if (static_cast<float>(max_thread_pair.v) < tmp_val) {
        max_thread_pair.set(static_cast<T>(tmp_val), i);
      }
#ifdef DEBUG_TOPP
      if (i < 10) {
        printf(
            "tid: %d, i: %d, random_ratio: %f, thread_count: %f, tmp_val: %f, "
            "max_thread_pair.v: %f, max_thread_pair.id: %d\n",
            tid,
            i,
            random_ratio,
            thread_count,
            tmp_val,
            max_thread_pair.v,
            static_cast<int>(max_thread_pair.id));
      }
#endif
    }
#ifdef DEBUG_TOPP
    printf("tid: %d, thread_count: %f, thread_offset: %f\n",
           tid,
           thread_count,
           thread_offset);
#endif
#ifdef PADDLE_WITH_HIP
    uint64_t activate_mask = __ballot(p_t <= thread_offset);
#else
    uint32_t activate_mask = __ballot_sync(FINAL_MASK, p_t <= thread_offset);
#endif

    i_activate = i;
    if (activate_mask != 0) {
      if (lane_id == 0) {
        atomicAdd(&stop_shared, 1);
      }
    }
    __syncthreads();
    if (stop_shared > 0) {
      break;
    }
  }
  __syncthreads();
  if (stop_shared == 0) {
    if (tid == 0) {
      out_id[bid] = sorted_id[offset];
      out_val[bid] = sorted_probs[offset];
    }
    return;
  }

  Pair<T> max_pair = BlockReduce(temp_storage_reduce)
                         .Reduce(max_thread_pair, MaxOp<Pair<T>>());
  if (tid == 0) {
    if (max_pair.id == -1) {
      max_pair.id = 0;
    }
#ifdef DEBUG_TOPP
    printf("max_id: %d, max_val: %f\n",
           static_cast<int>(max_pair.id),
           static_cast<float>(max_pair.v));
#endif
    out_id[bid] = sorted_id[offset + max_pair.id];
    out_val[bid] = sorted_probs[offset + max_pair.id];
  }
}

template <typename T, int BLOCK_SIZE>
__global__ void topp_sampling_ft(T* sorted_probs,
                                 int64_t* sorted_id,
                                 T* out_val,
                                 int64_t* out_id,
                                 const T* top_ps,
                                 const T* threshold,
                                 GPU(randState_t) * states,
                                 const int p_num,
                                 const int vocab_size,
                                 const bool need_batch_random,
                                 int* count_iter,
                                 int* count_iter_begin) {
  __shared__ int stop_shared;
  __shared__ float rand_p;
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  constexpr int NUM_WARPS = BLOCK_SIZE / 32;
  const int lane_id = tid % 32;
  const int warp_id = tid / 32;
  const float p_t = static_cast<float>(top_ps[bid]);
  const float threshold_now =
      threshold ? static_cast<float>(threshold[bid]) : 0.f;
  if (tid == 0) {
    stop_shared = 0;
    rand_p = p_t;
#ifdef DEBUG_TOPP
    printf("bi: %d, p: %f\n", bid, rand_p);
#endif
  }
  if (count_iter_begin[bid] == count_iter[bid + 1]) {
    // topk
    return;
  }

  typedef hipcub::BlockScan<float, BLOCK_SIZE> BlockScan;
  typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduce;
  __shared__ typename BlockScan::TempStorage temp_storage;
  __shared__ typename BlockReduce::TempStorage temp_storage_reduce;
#ifdef PADDLE_WITH_HIP
  __shared__ uint64_t selected_shared[NUM_WARPS];
#else
  __shared__ uint32_t selected_shared[NUM_WARPS];
#endif
  int threshold_id = 0;

  // Initialize running total
  BlockPrefixCallbackOp prefix_op(0);

  if (lane_id == 0) {
    selected_shared[warp_id] = 0;
  }
  __syncthreads();

  int offset = bid * vocab_size;
#ifdef DEBUG_TOPP
  if (tid == 0) {
    printf(
        "first_elem1_1: %f, first_elem1_2: %f, first_id1_1: %d, first_id1_2: "
        "%d\n",
        static_cast<float>(sorted_probs[offset]),
        static_cast<float>(sorted_probs[offset + 1]),
        static_cast<int>(sorted_id[offset]),
        static_cast<int>(sorted_id[offset + 1]));
  }
#endif
  int end = ((vocab_size + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
  int i_activate = 0;
  float thread_offset = 0;
  for (int i = tid; i < end; i += BLOCK_SIZE) {
    float thread_count =
        (i < vocab_size) ? static_cast<float>(sorted_probs[offset + i]) : 0.f;
    if (i < vocab_size && thread_count >= threshold_now) {
      threshold_id = i;
    }
    BlockScan(temp_storage)
        .InclusiveSum(thread_count, thread_offset, prefix_op);

#ifdef PADDLE_WITH_HIP
    uint64_t activate_mask = __ballot(rand_p <= thread_offset);
#else
    uint32_t activate_mask = __ballot_sync(FINAL_MASK, rand_p <= thread_offset);
#endif

    i_activate = i;
    if (activate_mask != 0) {
      if (lane_id == 0) {
        atomicAdd(&stop_shared, 1);
        selected_shared[warp_id] = activate_mask;
      }
    }
    __syncthreads();
    if (stop_shared > 0) {
      break;
    }
  }
  __syncthreads();
  if (stop_shared == 0) {
    if (tid == 0) {
      out_id[bid] = sorted_id[offset];
      out_val[bid] = sorted_probs[offset];
#ifdef DEBUG_TOPP
      printf("stop_shared: %d, out_id: %d, out_val: %f\n",
             static_cast<int>(stop_shared),
             static_cast<int>(out_id[bid]),
             static_cast<float>(out_val[bid]));
#endif
    }
    return;
  }
#ifdef DEBUG_TOPP
  if (tid == 0) {
    printf(
        "first_elem2_1: %f, first_elem2_2: %f, first_id2_1: %d, first_id2_2: "
        "%d\n",
        static_cast<float>(sorted_probs[offset]),
        static_cast<float>(sorted_probs[offset + 1]),
        static_cast<int>(sorted_id[offset]),
        static_cast<int>(sorted_id[offset + 1]));
  }
#endif
  bool skip = (selected_shared[warp_id] > 0) ? false : true;
  for (int i = 0; i < warp_id; i++) {
    if (selected_shared[i] != 0) {
      // If the previous has stopped, skip the current warp
      skip = true;
    }
  }
  if (!skip) {
    int active_lane_id =
        WARP_SIZE - __popc(selected_shared[warp_id]);  // first not 0
    if (lane_id == active_lane_id) {
      float val = static_cast<float>(sorted_probs[offset + i_activate]);
#ifdef DEBUG_TOPP
      printf(
          "active_lane_id: %d, i_activate: %d.\n", active_lane_id, i_activate);
      for (int i = 0; i < active_lane_id; i++) {
        printf("p %d, value: %f\n",
               i,
               static_cast<float>(sorted_probs[offset + i]));
      }
#endif
      if (val < threshold_now) {
        // don't sample low score token
        int max_id =
            BlockReduce(temp_storage_reduce).Reduce(threshold_id, MaxOp<int>());
#ifdef PADDLE_WITH_HIP
        hiprandStatePhilox4_32_10_t rng;
        hiprand_init(bid * blockDim.x + tid, tid, 0, &rng);
        int random_id = hiprand(&rng) % (max_id + 1);
#else
        hiprandStatePhilox4_32_10_t rng;
        hiprand_init(bid * blockDim.x + tid, tid, 0, &rng);
        int random_id = hiprand(&rng) % (max_id + 1);
#endif
        out_id[bid] = sorted_id[offset + random_id];
        out_val[bid] = sorted_probs[offset + random_id];
      } else {
        out_id[bid] = sorted_id[offset + i_activate];
        out_val[bid] = sorted_probs[offset + i_activate];
      }
    }
  }
}

template <typename T, typename Context>
void DispatchTopPSampling(const Context& dev_ctx,
                          T* sorted_probs,
                          int64_t* sorted_id,
                          T* out_val,
                          int64_t* out_id,
                          const T* top_ps,
                          const T* threshold,
                          GPU(randState_t) * states,
                          const int p_num,
                          const int vocab_size,
                          const int bs,
                          const bool need_batch_random,
                          int* count_iter,
                          int* count_iter_begin,
                          const std::string& mode) {
  int BlockSize = GetBlockSize(vocab_size);
  if (mode == "truncate") {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          topp_sampling_ft<T, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(sorted_probs,
                                                   sorted_id,
                                                   out_val,
                                                   out_id,
                                                   top_ps,
                                                   threshold,
                                                   states,
                                                   p_num,
                                                   vocab_size,
                                                   need_batch_random,
                                                   count_iter,
                                                   count_iter_begin));
      default:
        PD_THROW("the input data shape has error in the topp_sampling kernel.");
    }
  } else {
    switch (BlockSize) {
      FIXED_BLOCK_DIM(
          topp_sampling<T, kBlockDim>
          <<<bs, kBlockDim, 0, dev_ctx.stream()>>>(sorted_probs,
                                                   sorted_id,
                                                   out_val,
                                                   out_id,
                                                   top_ps,
                                                   threshold,
                                                   states,
                                                   p_num,
                                                   vocab_size,
                                                   need_batch_random,
                                                   count_iter,
                                                   count_iter_begin));
      default:
        PD_THROW("the input data shape has error in the topp_sampling kernel.");
    }
  }
}

__global__ void setup_kernel(GPU(randState_t) * state,
                             int64_t* seed,
                             const int bs) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < bs; i += gridDim.x * blockDim.x) {
#ifdef PADDLE_WITH_HIP
    hiprand_init(static_cast<uint64_t>(seed[i]), 0, 0, &state[i]);
#else
    hiprand_init(static_cast<uint64_t>(seed[i]), 0, 0, &state[i]);
#endif
  }
}

__global__ void setup_kernel(GPU(randState_t) * state,
                             const uint64_t seed,
                             const uint64_t offset,
                             const int bs,
                             const bool need_batch_random) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < bs; i += gridDim.x * blockDim.x) {
    if (need_batch_random) {
#ifdef PADDLE_WITH_HIP
      hiprand_init(seed, i, offset, &state[i]);
#else
      hiprand_init(seed, i, offset, &state[i]);
#endif
    } else {
#ifdef PADDLE_WITH_HIP
      hiprand_init(seed, 0, offset, &state[i]);
#else
      hiprand_init(seed, 0, offset, &state[i]);
#endif
    }
  }
}

#ifdef PADDLE_WITH_HIP
template <typename T>
__global__ void print_kernel(T* input, int size) {
  for (int i = 0; i < size; i++) {
    printf("[");
    if (i != size - 1) {
      printf("%f, ", static_cast<float>(input[i]));
    } else {
      printf("%f]\n", static_cast<float>(input[i]));
    }
  }
}
#else
template <typename T>
__global__ void print_kernel(T* input, int size) {
  for (int i = 0; i < size; i++) {
    std::stringstream ss;
    ss << "[";
    if (i != size - 1) {
      ss << static_cast<float>(input[i]) << ", ";
    } else {
      ss << static_cast<float>(input[i]) << "]\n";
    }
    VLOG(0) << ss.str();
  }
}
#endif

template <typename T>
T* SafeGetTensorPtr(const DenseTensor& t) {
  return const_cast<T*>(t.data<T>());
}

template <typename T>
T* SafeGetTensorPtr(const DenseTensor* t) {
  return t ? SafeGetTensorPtr<T>(*t) : nullptr;
}

template <typename T>
T* SafeGetTensorPtr(const paddle::optional<DenseTensor>& t) {
  return t ? SafeGetTensorPtr<T>(t.get()) : nullptr;
}

template <typename T, typename Context>
void TopPSamplingKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& ps,
                        const paddle::optional<DenseTensor>& threshold,
                        const paddle::optional<DenseTensor>& topp_seed,
                        int seed,
                        int k,
                        const std::string& mode,
                        DenseTensor* out,
                        DenseTensor* ids,
                        DenseTensor* topk_scores,
                        DenseTensor* topk_ids) {
  typedef DataTypeTraits<T> traits_;
  typedef typename traits_::DataType DataType_;
  auto cu_stream = dev_ctx.stream();
  const auto* input = &x;
  // get the input dims
  const auto& in_dims = input->dims();
  int p_num = ps.numel();
  int bs = in_dims[0];
  int vocab_size = in_dims[1];
  T* out_ptr = dev_ctx.template Alloc<T>(out);
  int64_t* ids_ptr = dev_ctx.template Alloc<int64_t>(ids);
  T* topk_scores_data = nullptr;
  int64_t* topk_ids_data = nullptr;
  if (k > 0) {
    topk_scores_data = dev_ctx.template Alloc<T>(topk_scores);
    topk_ids_data = dev_ctx.template Alloc<int64_t>(topk_ids);
  }

  DenseTensor ps_now;
  ps_now.Resize(phi::make_ddim({bs, 1}));
  dev_ctx.template Alloc<T>(&ps_now);
  phi::Copy(dev_ctx, ps, dev_ctx.GetPlace(), false, &ps_now);

  DenseTensor inds_input;
  inds_input.Resize(phi::make_ddim({bs, vocab_size}));
  dev_ctx.template Alloc<int64_t>(&inds_input);

  DenseTensor sorted_out;
  sorted_out.Resize(phi::make_ddim({bs, vocab_size}));
  dev_ctx.template Alloc<T>(&sorted_out);

  DenseTensor sorted_id;
  sorted_id.Resize(phi::make_ddim({bs, vocab_size}));
  dev_ctx.template Alloc<int64_t>(&sorted_id);

  int BlockSize = GetBlockSize(vocab_size);

  switch (BlockSize) {
    FIXED_BLOCK_DIM(FillIndex<int64_t><<<bs, kBlockDim, 0, cu_stream>>>(
        inds_input.data<int64_t>(), bs, vocab_size));
    default:
      PD_THROW("the input data shape has error in the FillIndex kernel.");
  }
  int64_t* infer_seed = SafeGetTensorPtr<int64_t>(topp_seed);

  GPU(randState_t) * states{nullptr};
  phi::Allocator::AllocationPtr rand_states_buf{nullptr};
  rand_states_buf = phi::memory_utils::Alloc(
      dev_ctx.GetPlace(),
      bs * sizeof(GPU(randState_t)),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  states = reinterpret_cast<GPU(randState_t)*>(rand_states_buf->ptr());

  uint64_t seed_now = seed;
  uint64_t offset = 0;
  bool need_batch_random = false;

  if (infer_seed) {
    setup_kernel<<<1, 256, 0, cu_stream>>>(states, infer_seed, bs);
  } else {
    if (seed == -1) {
      need_batch_random = true;
      auto gen_cuda = dev_ctx.GetGenerator();
      uint64_t increment = ps.numel() * 4;
      auto seed_offset = gen_cuda->IncrementOffset(increment);
      seed = seed_offset.first;
      offset = seed_offset.second;
      setup_kernel<<<1, 256, 0, cu_stream>>>(
          states, seed, offset, bs, need_batch_random);
    } else {
      setup_kernel<<<1, 256, 0, cu_stream>>>(
          states, seed, offset, bs, need_batch_random);
    }
  }

  DenseTensor count_iter;
  count_iter.Resize(phi::make_ddim({bs + 1}));
  dev_ctx.template Alloc<int>(&count_iter);
  DenseTensor count_iter_begin;
  count_iter_begin.Resize(phi::make_ddim({bs}));
  dev_ctx.template Alloc<int>(&count_iter_begin);
  SetCountIter<<<1, 256, 0, cu_stream>>>(count_iter.data<int>(), bs + 1);

  T* threshold_data = SafeGetTensorPtr<T>(threshold);

  constexpr int TopKMaxLength = 2;
  constexpr int TopPBeamTopK = 5;

  DispatchKeMatrixTopPBeamTopK<T, Context, TopKMaxLength, TopPBeamTopK>(
      dev_ctx,
      x.data<T>(),
      threshold_data,
      states,
      ps_now.data<T>(),
      ids_ptr,
      out_ptr,
      topk_ids_data,
      topk_scores_data,
      vocab_size,
      count_iter.data<int>(),
      count_iter_begin.data<int>(),
      k,
      bs,
      need_batch_random,
      mode);

  size_t temp_storage_bytes = 0;

  hipcub::TransformInputIterator<int, SegmentOffsetIter, int*>
      segment_offsets_t_begin(count_iter_begin.data<int>(),
                              SegmentOffsetIter(vocab_size));

  hipcub::TransformInputIterator<int, SegmentOffsetIter, int*>
      segment_offsets_t_end(count_iter.data<int>(),
                            SegmentOffsetIter(vocab_size));

  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr,
      temp_storage_bytes,
      reinterpret_cast<DataType_*>(const_cast<T*>(x.data<T>())),
      reinterpret_cast<DataType_*>(const_cast<T*>(sorted_out.data<T>())),
      inds_input.data<int64_t>(),
      sorted_id.data<int64_t>(),
      vocab_size * bs,
      bs,
      segment_offsets_t_begin,
      segment_offsets_t_end + 1,
      0,
      sizeof(T) * 8,
      cu_stream);

  temp_storage_bytes = div_up(temp_storage_bytes, 256) * 256;
  int64_t temp_size = temp_storage_bytes;
  DenseTensor temp_storage;
  temp_storage.Resize(phi::make_ddim({temp_size}));
  dev_ctx.template Alloc<uint8_t>(&temp_storage);

  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      temp_storage.data<uint8_t>(),
      temp_storage_bytes,
      reinterpret_cast<DataType_*>(const_cast<T*>(x.data<T>())),
      reinterpret_cast<DataType_*>(const_cast<T*>(sorted_out.data<T>())),
      inds_input.data<int64_t>(),
      sorted_id.data<int64_t>(),
      vocab_size * bs,
      bs,
      segment_offsets_t_begin,
      segment_offsets_t_end + 1,
      0,
      sizeof(T) * 8,
      cu_stream);

  DispatchTopPSampling<T>(dev_ctx,
                          sorted_out.data<T>(),
                          sorted_id.data<int64_t>(),
                          out_ptr,
                          ids_ptr,
                          ps_now.data<T>(),
                          threshold_data,
                          states,
                          p_num,
                          vocab_size,
                          bs,
                          need_batch_random,
                          count_iter.data<int>(),
                          count_iter_begin.data<int>(),
                          mode);
}

}  // namespace phi

PD_REGISTER_KERNEL(top_p_sampling,
                   GPU,
                   ALL_LAYOUT,
                   phi::TopPSamplingKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
