// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/nll_loss_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/nll_loss.h"

namespace phi {

template <typename T, typename Context>
void NllLossRawKernel(const Context& dev_ctx,
                      const DenseTensor& input,
                      const DenseTensor& label,
                      const paddle::optional<DenseTensor>& weight,
                      int64_t ignore_index,
                      const std::string& reduction,
                      DenseTensor* out,
                      DenseTensor* total_weight) {
  auto* x = &input;
  auto x_data = x->data<T>();
  auto out_data = dev_ctx.template Alloc<T>(out);
  auto total_weight_data = dev_ctx.template Alloc<T>(total_weight);
  auto label_data = label.data<int64_t>();
  auto weight_data = weight.get_ptr() ? weight.get_ptr()->data<T>() : nullptr;
#ifdef PADDLE_WITH_HIP
  hipMemset(total_weight_data, 0, sizeof(T));
#else
  hipMemset(total_weight_data, 0, sizeof(T));
#endif
  auto x_dims = x->dims();
  auto batch_size = x_dims[0];
  auto n_classes = x_dims[1];
  int64_t size_average = (int64_t)(reduction == "mean");

  if (x_dims.size() == 2) {
    int blocks = NumBlocks(batch_size);
    int threads = kNumCUDAThreads;
    if (reduction == "none") {
      GPUNLLLossForward1D_no_reduce<T>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(out_data,
                                                     x_data,
                                                     label_data,
                                                     weight_data,
                                                     batch_size,
                                                     n_classes,
                                                     ignore_index);
    } else {
      GPUNLLLossForward1D_with_reduce<T>
          <<<1, NTHREADS, 0, dev_ctx.stream()>>>(out_data,
                                                 total_weight_data,
                                                 x_data,
                                                 label_data,
                                                 weight_data,
                                                 batch_size,
                                                 n_classes,
                                                 size_average,
                                                 ignore_index);
    }
  } else if (x_dims.size() == 4) {
    const auto in_dim2 = x_dims[2];
    const auto in_dim3 = x_dims[3];
    const auto map_size = in_dim2 * in_dim3;
    const auto out_numel = batch_size * in_dim2 * in_dim3;
    int blocks = NumBlocks(out_numel);
    int threads = kNumCUDAThreads;
    if (reduction == "none") {
      GPUNLLLossForward2D_no_reduce<T>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(out_data,
                                                     x_data,
                                                     label_data,
                                                     weight_data,
                                                     batch_size,
                                                     n_classes,
                                                     in_dim2,
                                                     in_dim3,
                                                     ignore_index);
    } else {
      int blocks_per_sample = NumBlocks(map_size) / 128;
      blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
      int total_blocks = blocks_per_sample * batch_size;
      GPUNLLLossForward2D_with_reduce<T>
          <<<total_blocks, threads, 0, dev_ctx.stream()>>>(out_data,
                                                           total_weight_data,
                                                           x_data,
                                                           label_data,
                                                           weight_data,
                                                           batch_size,
                                                           n_classes,
                                                           map_size,
                                                           blocks_per_sample,
                                                           ignore_index);
      if (size_average) {
        GPUNLLLossForward2D_size_average<T>
            <<<1, 1, 0, dev_ctx.stream()>>>(out_data, total_weight_data);
      }
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(
    nll_loss, GPU, ALL_LAYOUT, phi::NllLossRawKernel, float, double) {}
