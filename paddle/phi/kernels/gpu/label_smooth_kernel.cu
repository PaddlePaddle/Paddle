#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <vector>
#include "paddle/fluid/operators/elementwise/elementwise_op_impl.cu.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/label_smooth_kernel.h"

namespace phi {

template <typename T>
struct LabelSmoothFunctor {
  T epsilon;
  T label_dim;

  __forceinline__ LabelSmoothFunctor(float epsilon_data, int label_dim_data) {
    epsilon = static_cast<T>(epsilon_data);
    label_dim = static_cast<T>(label_dim_data);
  }

  __device__ __forceinline__ T operator()(const T x) const {
    return (static_cast<T>(1 - epsilon) * x +
            static_cast<T>(epsilon / label_dim));
  }
};

template <typename T>
__global__ void LabelSmoothRunDistKernel(const int N,
                                         const float epsilon,
                                         const int dist_numel,
                                         const T* src,
                                         const T* dist_data,
                                         T* dst) {
  CUDA_KERNEL_LOOP(idx, N) {
    int dist_idx = idx % dist_numel;
    dst[idx] = static_cast<T>(1 - epsilon) * src[idx] +
               static_cast<T>(epsilon) * dist_data[dist_idx];
  }
}

template <typename T, typename Context>
void LabelSmoothKernel(const Context& ctx,
                       const DenseTensor& label,
                       paddle::optional<const DenseTensor&> prior_dist,
                       float epsilon,
                       DenseTensor* out) {
  auto label_dim = label.dims()[label.dims().size() - 1];
  auto size_prob = label.numel();
  const T* in_data = label.data<T>();
  T* out_data = ctx.template Alloc<T>(out);

  if (prior_dist.get_ptr()) {
    int threads = 512;
    int grid = (size_prob + threads - 1) / threads;
    auto stream = ctx.stream();
    const auto* dist_t = prior_dist.get_ptr();
    auto dist_numel = dist_t->numel();
    const T* dist_data = dist_t->data<T>();
    LabelSmoothRunDistKernel<T><<<grid, threads, 0, stream>>>(
        size_prob, epsilon, dist_numel, in_data, dist_data, out_data);

  } else {
    std::vector<const DenseTensor*> ins = {&label};
    std::vector<DenseTensor*> outs = {out};
    auto functor = LabelSmoothFunctor<T>(epsilon, label_dim);
    paddle::operators::LaunchSameDimsElementwiseCudaKernel<T>(
        ctx, ins, &outs, functor);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(
    label_smooth, GPU, ALL_LAYOUT, phi::LabelSmoothKernel, float, double) {}
