#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_device_function.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

namespace {

static inline int DivUp(int x, int y) { return (x + y - 1) / y; }

// Compute input gradient (shared memory version, for small future_context)
template <typename T>
__global__ void RowConvGradInputSharedMemory(const T *dout,
                                             const T *wt,
                                             int num_sequence,
                                             int input_dim,
                                             int future_context,
                                             const size_t *batch_indices,
                                             T *din) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int d = blockIdx.x * blx + thx;  // index along input dim

  extern __shared__ T mem[];
  T *sw = mem;
  if (thy < future_context) {
    sw[thy * blx + thx] =
        (d < input_dim) ? wt[thy * input_dim + d] : static_cast<T>(0);
  }
  __syncthreads();

  int current_timesteps = 0;
  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    current_timesteps = end - start;

    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < future_context) && ((k - w) >= 0); w++) {
        sum += (d < input_dim)
                   ? (sw[w * blx + thx] * dout[(k + start - w) * input_dim + d])
                   : static_cast<T>(0);
      }
      if (d < input_dim) {
        din[(k + start) * input_dim + d] = sum;
      }
    }
  }
}

// Compute input gradient (Naive version)
template <typename T>
__global__ void RowConvGradInput(const T *dout,
                                 const T *wt,
                                 int num_sequence,
                                 int input_dim,
                                 int future_context,
                                 const size_t *batch_indices,
                                 T *din) {
  int d = blockIdx.x * blockDim.x + threadIdx.x;  // index along input_dim
  int bly = blockDim.y;
  int thy = threadIdx.y;

  if (d >= input_dim) return;
  int current_timesteps = 0;

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    current_timesteps = end - start;

    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < future_context) && ((k - w) >= 0); w++) {
        sum += (wt[w * input_dim + d] * dout[(k + start - w) * input_dim + d]);
      }
      din[(k + start) * input_dim + d] = sum;
    }
  }
}

// Compute W gradient (small future_context version)
template <typename T>
__global__ void RowConvGradFilterImproved(const T *in,
                                          const T *dout,
                                          int num_sequence,
                                          int input_dim,
                                          int future_context,
                                          int block_x,
                                          int block_y,
                                          const size_t *batch_indices,
                                          T *dfilter) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int gx = blockIdx.x * blx;
  int d = gx + thx;  // index along input dim

  extern __shared__ T mem[];

  int xdim_sh_in = block_y;
  int xdim_sh_dout = block_y;
  int ydim_sh_in = block_x;
  int ydim_sh_dout = block_x + future_context - 1;
  int ydim_sh_dfilter = block_y;

  T *sh_in = mem;
  T *sh_dout = &mem[xdim_sh_in * ydim_sh_in];
  T *sh_dfilter = &mem[xdim_sh_in * ydim_sh_in + xdim_sh_dout * ydim_sh_dout];

  if (thy < future_context) {
    sh_dfilter[thy * ydim_sh_dfilter + thx] = static_cast<T>(0);
  }
  __syncthreads();

  // NOTE(zcd): temporary solution
  unsigned mask = 0u;
  CREATE_SHFL_MASK(mask, true);

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;

    int scaled_cur_steps =
        ((current_timesteps + block_x - 1) / block_x) * block_x;

    for (int k = thy; k < scaled_cur_steps; k += block_x) {
      int pos = start + k;
      sh_in[thx * ydim_sh_in + thy] =
          (d < input_dim && pos < end) ? in[pos * input_dim + d] : T(0);
      sh_dout[thx * ydim_sh_dout + thy + future_context - 1] =
          (d < input_dim && pos < end) ? dout[pos * input_dim + d] : T(0);
      __syncthreads();

      if (thy < future_context - 1) {
        int pos_offset = pos - future_context + 1;
        sh_dout[thx * ydim_sh_dout + thy] =
            (d < input_dim && pos_offset >= start)
                ? dout[pos_offset * input_dim + d]
                : T(0);
      }
      __syncthreads();

      for (int w = 0; w < future_context; w++) {
        T val = sh_in[thy * ydim_sh_in + thx] *
                sh_dout[thy * ydim_sh_dout + thx + future_context - 1 - w];
        __syncthreads();

        for (int offset = 16; offset > 0;
             offset = offset / 2) {  // blockDim.x is 32.
          val += phi::backends::gpu::CudaShuffleDownSync(mask, val, offset);
        }
        __syncthreads();

        if (thx == 0) {
          sh_dfilter[w * ydim_sh_dfilter + thy] += val;
        }
        __syncthreads();
      }
    }
  }
  for (int w = thy; (w < future_context) && (d < input_dim); w += bly) {
    dfilter[w * input_dim + d] += sh_dfilter[w * ydim_sh_dfilter + thx];
  }
}

// Compute weight(filter) gradient
template <typename T>
__global__ void RowConvGradFilter(const T *in,
                                  const T *dout,
                                  int num_sequence,
                                  int input_dim,
                                  int future_context,
                                  int block_x,
                                  int block_y,
                                  const size_t *batch_indices,
                                  T *dfilter) {
  int blx = blockDim.x;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int gx = blockIdx.x * blx;
  int d = gx + thx;  // index along input dim
  extern __shared__ T mem[];
  T *sh_in = mem;
  T *sh_dout = &mem[block_x * block_y];

  // NOTE(zcd): temporary solution
  unsigned mask = 0u;
  CREATE_SHFL_MASK(mask, true);
  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;

    int scaled_cur_steps =
        ((current_timesteps + block_x - 1) / block_x) * block_x;

    for (int k = thy; k < scaled_cur_steps; k += block_x) {
      int pos = start + k;
      sh_in[thx * block_y + thy] =
          (d < input_dim && pos < end) ? in[pos * input_dim + d] : 0.0;
      __syncthreads();

      for (int w = 0; w < future_context; w++) {
        sh_dout[thx * block_y + thy] =
            (d < input_dim && (k - w) >= 0 && (k - w) < current_timesteps)
                ? dout[(pos - w) * input_dim + d]
                : 0.0;
        __syncthreads();

        T val = sh_in[thy * block_y + thx] * sh_dout[thy * block_y + thx];
        __syncthreads();

        for (int offset = 16; offset > 0;
             offset = offset / 2) {  // blockDim.x is 32.
          val += phi::backends::gpu::CudaShuffleDownSync(mask, val, offset);
        }
        __syncthreads();

        if (thx == 0 && (gx + thy) < input_dim) {
          dfilter[w * input_dim + gx + thy] += val;
        }
      }
    }
  }
}

}  // namespace

template <typename T, typename Context>
void RowConvGradKernel(const Context &dev_ctx,
                       const DenseTensor &x_in,
                       const DenseTensor &filter_in,
                       const DenseTensor &out_grad,
                       DenseTensor *x_grad,
                       DenseTensor *filter_grad) {
  auto *X = &x_in;
  auto *Filter = &filter_in;
  auto *dOut = &out_grad;
  const T *in = X->data<T>();
  const T *weights = Filter->data<T>();
  const T *dout = dOut->data<T>();

  phi::DenseTensor *dX = x_grad;
  phi::DenseTensor *dFilter = filter_grad;
  int batch_size = 0;
  bool is_tensor = X->lod().empty();
  if (is_tensor) {
    batch_size = X->dims()[0];
  } else {
    batch_size = X->lod()[0].size() - 1;
  }

  int input_dim = 0;
  phi::Vector<size_t> batch_indices(batch_size + 1);
  int timesteps = X->dims()[1];
  if (is_tensor) {
    for (int i = 0; i < batch_size + 1; i++) {
      batch_indices[i] = i * timesteps;
    }
    input_dim = X->dims()[2];
  } else {
    batch_indices = X->lod()[0];
    input_dim = X->dims()[1];
  }
  // int input_dim = X->dims()[1];
  int num_sequence = batch_indices.size() - 1;
  int future_context = Filter->dims()[0];
  phi::MixVector<size_t> mixv_batch_indices(&batch_indices);
  size_t *idx = mixv_batch_indices.CUDAMutableData(dev_ctx.GetPlace());

  auto &device_ctx = dev_ctx;
  phi::funcs::SetConstant<phi::GPUContext, T> zero;

  if (dFilter) {
    T *dfilter = dev_ctx.template Alloc<T>(dFilter);
    zero(device_ctx, dFilter, static_cast<T>(0.0));

    if (future_context <= 32) {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      int block_x = block_dim.x;
      int block_y = block_dim.y;
      int mem_per_block =
          (block_y * block_x + block_y * (block_x + future_context - 1) +
           future_context * block_y) *
          sizeof(T);
      RowConvGradFilterImproved<T>
          <<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
              in,
              dout,
              num_sequence,
              input_dim,
              future_context,
              block_x,
              block_y,
              idx,
              dfilter);
    } else {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      int block_x = block_dim.x;
      int block_y = block_dim.y;
      int mem_per_block =
          (block_x * block_y * 2) * sizeof(T);  // For 2 arrays of size 32x32
      RowConvGradFilter<T>
          <<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
              in,
              dout,
              num_sequence,
              input_dim,
              future_context,
              block_x,
              block_y,
              idx,
              dfilter);
    }
  }

  if (dX) {
    T *din = dev_ctx.template Alloc<T>(dX);
    if (future_context <= 32) {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      int mem_per_block = (future_context * block_dim.x) * sizeof(T);
      RowConvGradInputSharedMemory<T>
          <<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
              dout, weights, num_sequence, input_dim, future_context, idx, din);
    } else {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      RowConvGradInput<T><<<grid_dim, block_dim, 0, device_ctx.stream()>>>(
          dout, weights, num_sequence, input_dim, future_context, idx, din);
    }
  }
  mixv_batch_indices.CopyToCPU();
}
}  // namespace phi

PD_REGISTER_KERNEL(
    row_conv_grad, GPU, ALL_LAYOUT, phi::RowConvGradKernel, float) {}
