#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/bincount_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename InputT, typename OutT>
__global__ void KernelBincount(const InputT* input,
                               const int total_elements,
                               const bool has_weights,
                               const T* weights,
                               OutT* output) {
  if (!has_weights) {
    for (int i = threadIdx.x; i < total_elements; i += blockDim.x) {
      paddle::platform::CudaAtomicAdd(&output[input[i]], 1L);
    }
  } else {
    for (int i = threadIdx.x; i < total_elements; i += blockDim.x) {
      paddle::platform::CudaAtomicAdd(&output[input[i]],
                                      static_cast<OutT>(weights[i]));
    }
  }
}

template <typename Context, typename T, typename InputT>
void BincountCUDAInner(const Context& dev_ctx,
                       const DenseTensor& x,
                       const paddle::optional<DenseTensor>& weights,
                       int minlength,
                       DenseTensor* out) {
  const DenseTensor* input = &x;
  DenseTensor* output = out;
  const InputT* input_data = input->data<InputT>();

  const int input_numel = input->numel();

  if (input_data == nullptr) {
    phi::DDim out_dim{0};
    output->Resize(out_dim);
    dev_ctx.template Alloc<T>(output);
    return;
  }
  auto input_x = EigenVector<InputT>::Flatten(*input);
  DenseTensor input_min_t, input_max_t;
  input_max_t.Resize({1});
  auto* input_max_data = dev_ctx.template Alloc<InputT>(&input_max_t);
  input_min_t.Resize({1});
  auto* input_min_data = dev_ctx.template Alloc<InputT>(&input_min_t);

  auto input_max_scala = EigenScalar<InputT>::From(input_max_t);
  auto input_min_scala = EigenScalar<InputT>::From(input_min_t);

  auto* place = dev_ctx.eigen_device();
  input_max_scala.device(*place) = input_x.maximum();
  input_min_scala.device(*place) = input_x.minimum();

  DenseTensor input_min_cpu, input_max_cpu;
  paddle::framework::TensorCopySync(
      input_max_t, phi::CPUPlace(), &input_max_cpu);
  paddle::framework::TensorCopySync(
      input_min_t, phi::CPUPlace(), &input_min_cpu);

  InputT input_min = input_min_cpu.data<InputT>()[0];

  PADDLE_ENFORCE_GE(
      input_min,
      static_cast<InputT>(0),
      phi::errors::InvalidArgument(
          "The elements in input tensor must be non-negative ints"));

  int64_t output_size =
      static_cast<int64_t>(input_max_cpu.data<InputT>()[0]) + 1L;

  output_size = std::max(output_size, static_cast<int64_t>(minlength));
  phi::DDim out_dim{output_size};
  output->Resize(out_dim);

  bool has_weights = weights.is_initialized();

  const T* weights_data = has_weights ? weights->data<T>() : nullptr;
  auto stream = dev_ctx.stream();

  if (!has_weights) {
    int64_t* output_data = dev_ctx.template Alloc<int64_t>(output);
    phi::funcs::SetConstant<Context, int64_t>()(dev_ctx, output, 0L);

    KernelBincount<T, InputT, int64_t>
        <<<GET_BLOCKS(input_numel), PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
            input_data, input_numel, has_weights, weights_data, output_data);
  } else {
    const auto& weights_type =
        paddle::framework::TransToProtoVarType(weights->dtype());

    if (weights->dtype() == DataType::FLOAT32) {
      float* output_data = dev_ctx.template Alloc<float>(output);
      phi::funcs::SetConstant<Context, float>()(
          dev_ctx, output, static_cast<float>(0));

      KernelBincount<T, InputT, float>
          <<<GET_BLOCKS(input_numel), PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
              input_data, input_numel, has_weights, weights_data, output_data);
    } else {
      double* output_data = dev_ctx.template Alloc<double>(output);
      phi::funcs::SetConstant<Context, double>()(
          dev_ctx, output, static_cast<double>(0));
      KernelBincount<T, InputT, double>
          <<<GET_BLOCKS(input_numel), PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
              input_data, input_numel, has_weights, weights_data, output_data);
    }
  }
}

template <typename T, typename Context>
void BincountKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const paddle::optional<DenseTensor>& weights,
                    const Scalar& minlength,
                    DenseTensor* out) {
  int int_minlength = minlength.to<int>();
  PADDLE_ENFORCE_GE(int_minlength,
                    0,
                    phi::errors::InvalidArgument(
                        "The minlength should be greater than or equal to 0."
                        "But received minlength is %d",
                        int_minlength));

  if (x.dtype() == DataType::INT32) {
    BincountCUDAInner<Context, T, int>(dev_ctx, x, weights, int_minlength, out);
  } else if (x.dtype() == DataType::INT64) {
    BincountCUDAInner<Context, T, int64_t>(
        dev_ctx, x, weights, int_minlength, out);
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(bincount,
                   GPU,
                   ALL_LAYOUT,
                   phi::BincountKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
