#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/bce_loss_grad_kernel.h"

#include <algorithm>
#include <vector>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/hostdevice.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"

namespace phi {

template <typename T>
struct BCELossGradFunctor {
  T one;
  T eps;

  HOSTDEVICE inline BCELossGradFunctor() {
    one = static_cast<T>(1.0f);
    eps = static_cast<T>(1e-12);
  }

  HOSTDEVICE inline T operator()(const T x, const T label, const T dout) const {
    T term1 = max((one - x) * x, eps);
    return (dout * (x - label) / term1);
  }
};

template <typename T, typename Context>
void BCELossGradKernel(const Context& dev_ctx,
                       const DenseTensor& input,
                       const DenseTensor& label,
                       const DenseTensor& out_grad,
                       DenseTensor* input_grad) {
  dev_ctx.template Alloc<T>(input_grad);
  std::vector<const DenseTensor*> ins = {&input, &label, &out_grad};
  std::vector<DenseTensor*> outs = {input_grad};
  auto functor = BCELossGradFunctor<T>();
  phi::funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    bce_loss_grad, GPU, ALL_LAYOUT, phi::BCELossGradKernel, float, double) {}
