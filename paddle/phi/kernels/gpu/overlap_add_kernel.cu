// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/overlap_add_kernel.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/overlap_add_functor.h"

namespace phi {

template <typename T, typename Context>
void OverlapAddKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      int hop_length,
                      int axis,
                      DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  const size_t x_rank = x.dims().size();
  const size_t out_rank = out->dims().size();

  const int n_frames = (axis == 0) ? x.dims()[0] : x.dims()[x_rank - 1];
  const int frame_length = (axis == 0) ? x.dims()[1] : x.dims()[x_rank - 2];
  const int seq_length =
      (axis == 0) ? out->dims()[0] : out->dims()[out_rank - 1];

  // auto& dev_ctx = ctx.device_context<Context>();

  DenseTensor x_(x.type());
  x_ = x;

  phi::DDim preserved_dims;
  if (out_rank > 2) {
    // Save dims used to flatten both input and output tensors and restore
    // output tensor.
    phi::DDim x_resized_dims;
    phi::DDim out_resized_dims;
    if (axis == 0) {
      preserved_dims = phi::slice_ddim(out->dims(), 1, out_rank);
      x_resized_dims = {n_frames, frame_length, phi::product(preserved_dims)};
      out_resized_dims = {seq_length, phi::product(preserved_dims)};
    } else {
      preserved_dims = phi::slice_ddim(out->dims(), 0, out_rank - 1);
      x_resized_dims = {phi::product(preserved_dims), frame_length, n_frames};
      out_resized_dims = {phi::product(preserved_dims), seq_length};
    }
    x_.Resize(x_resized_dims);
    out->Resize(out_resized_dims);
  }

  DenseTensor trans_x(x_.type());
  DenseTensor trans_out(out->type());

  // Transpose input and output in case that axis is 0.
  if (axis == 0) {
    if (out_rank == 1U) {
      trans_out = *out;

      std::vector<int> perm_x{1, 0};
      auto x_dims_vec = phi::vectorize(x_.dims());
      for (int i = 0; i < x_.dims().size(); ++i) {
        x_dims_vec[i] = x_.dims()[perm_x[i]];
      }
      trans_x.Resize(phi::make_ddim(x_dims_vec));
      dev_ctx.template Alloc<T>(&trans_x);
      phi::funcs::TransCompute<Context, T>(
          perm_x.size(), dev_ctx, x_, &trans_x, perm_x);
    } else {
      std::vector<int> perm_out{1, 0};
      auto out_dims_vec = phi::vectorize(out->dims());
      for (int i = 0; i < out->dims().size(); ++i) {
        out_dims_vec[i] = out->dims()[perm_out[i]];
      }
      trans_out.Resize(phi::make_ddim(out_dims_vec));
      dev_ctx.template Alloc<T>(&trans_out);
      phi::funcs::TransCompute<Context, T>(
          perm_out.size(), dev_ctx, *out, &trans_out, perm_out);

      std::vector<int> perm_x{2, 1, 0};
      auto x_dims_vec = phi::vectorize(x_.dims());
      for (int i = 0; i < x_.dims().size(); ++i) {
        x_dims_vec[i] = x_.dims()[perm_x[i]];
      }
      trans_x.Resize(phi::make_ddim(x_dims_vec));
      dev_ctx.template Alloc<T>(&trans_x);
      phi::funcs::TransCompute<Context, T>(
          perm_x.size(), dev_ctx, x_, &trans_x, perm_x);
    }
  } else {
    trans_x = x_;
    trans_out = *out;
  }

  OverlapAddFunctor<Context, T>()(dev_ctx,
                                  &trans_x,
                                  &trans_out,
                                  seq_length,
                                  frame_length,
                                  n_frames,
                                  hop_length,
                                  /*is_grad*/ false);

  // Transpose output in case axis is 0.
  if (axis == 0 && out_rank > 1U) {
    std::vector<int> perm_out{1, 0};
    phi::funcs::TransCompute<Context, T>(
        perm_out.size(), dev_ctx, trans_out, out, perm_out);
  }

  // Restore output dims when the number of dims is larger than 2.
  if (out_rank > 2) {
    std::vector<int64_t> restored_out_shape;
    for (int i = 0; i < preserved_dims.size(); i++) {
      restored_out_shape.push_back(preserved_dims[i]);
    }

    if (axis == 0) {
      // (seq_length, ...)
      restored_out_shape.insert(restored_out_shape.begin(), seq_length);
    } else {
      // (..., seq_length)
      restored_out_shape.push_back(seq_length);
    }

    out->Resize(phi::make_ddim(restored_out_shape));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(overlap_add,
                   GPU,
                   ALL_LAYOUT,
                   phi::OverlapAddKernel,
                   int,
                   int64_t,
                   float,
                   double,
                   paddle::platform::float16,
                   paddle::platform::complex<float>,
                   paddle::platform::complex<double>) {}
