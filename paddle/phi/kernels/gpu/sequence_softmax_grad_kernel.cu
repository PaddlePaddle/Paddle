#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif

#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math.h"
#include "paddle/phi/kernels/impl/sequence_softmax_kernel_impl.h"

namespace phi {

template <typename T, int BlockDim>
using BlockReduce = hipcub::BlockReduce<T, BlockDim>;

template <typename T, int BlockDim>
using BlockReduceTempStorage = typename BlockReduce<T, BlockDim>::TempStorage;

template <typename T, int BlockDim>
__global__ void sequence_softmax_grad_kernel(const T *softmax_grad_data,
                                             const T *softmax_data,
                                             const size_t *ref_lod,
                                             const size_t src_hight,
                                             T *dx_data) {
  __shared__ BlockReduceTempStorage<T, BlockDim> temp_storage;
  __shared__ T shared_data;

  for (int i = blockIdx.x; i < src_hight; i += gridDim.x) {
    size_t start = ref_lod[i];
    size_t span = ref_lod[i + 1] - start;

    T result = 0;
    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      size_t idx = start + tid;
      T s_g_d = softmax_grad_data[idx];
      T s_d = softmax_data[idx];
      result += s_g_d * s_d;
    }
    result = BlockReduce<T, BlockDim>(temp_storage).Reduce(result, hipcub::Sum());
    if (threadIdx.x == 0) {
      shared_data = result;
    }
    __syncthreads();

    for (int tid = threadIdx.x; tid < span; tid += blockDim.x) {
      size_t idx = start + tid;
      T s_g_d = softmax_grad_data[idx];
      T s_d = softmax_data[idx];
      dx_data[idx] = (s_g_d - shared_data) * s_d;
    }
  }
}

template <typename T>
struct SequenceSoftmaxGradFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext &context,
                  const DenseTensor &dout,
                  const DenseTensor &out,
                  const phi::Vector<size_t> &ref_lod, /*referenced lod*/
                  DenseTensor *dx) {
    size_t height = ref_lod.size() - 1;

    const int kThreadsPerBlock = 32;
    int thread_x = kThreadsPerBlock;
    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);

    dim3 block_size(thread_x);
    dim3 grid_size(max_blocks);

    phi::MixVector<size_t> mixv_ref_lod(&ref_lod);
    sequence_softmax_grad_kernel<T, kThreadsPerBlock>
        <<<grid_size, block_size, 0, context.stream()>>>(
            dout.data<T>(),
            out.data<T>(),
            mixv_ref_lod.CUDAData(context.GetPlace()),
            height,
            context.Alloc<T>(dx));
  }
};

}  // namespace phi

PD_REGISTER_KERNEL(sequence_softmax_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SequenceSoftmaxGradKernel,
                   float,
                   double) {}
