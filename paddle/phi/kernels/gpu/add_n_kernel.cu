#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/add_n_kernel.h"

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

template <class T>
__global__ void SumArrayCUDAKernel(
    T **in, T *out, int64_t N, size_t in_size, bool read_dst) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    T total(read_dst ? out[id] : static_cast<T>(0));
    for (int i = 0; i < in_size; ++i) {
      const T *tmp = in[i];
      if (tmp) {
        total += tmp[id];
      }
    }
    out[id] = total;
    id += blockDim.x * gridDim.x;
  }
}

template <typename T, typename Context>
void AddNKernel(const Context &dev_ctx,
                const std::vector<const DenseTensor *> &x,
                DenseTensor *out) {
  const size_t in_num = x.size();

  constexpr size_t theory_sm_threads = 1024;
  auto stream = dev_ctx.stream();

  auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  auto sm_count = max_threads / theory_sm_threads;
  size_t tile_size = 0;
  dim3 grids;
  dim3 blocks;

  auto ComputeKernelParameter = [&](size_t length) {
    if (length >= max_threads)
      tile_size = 1024;
    else if (length < max_threads && length > sm_count * 128)
      tile_size = 512;
    else if (length <= sm_count * 128)
      tile_size = 256;
    grids = dim3(CEIL_DIV(length, tile_size), 1, 1);
    blocks = dim3(tile_size, 1, 1);
  };

  bool in_place = x[0] == out;

  if (!in_place) {
    auto *out_ptr = dev_ctx.template Alloc<T>(out);
    if (in_num >= 1) {
      auto &in_0_tensor = *x[0];
      if (in_0_tensor.numel() > 0) {
        in_place = (in_0_tensor.data<T>() == out_ptr);
      }
    }
  }

  // Sum of two tensors
  if (in_num == 2) {
    auto &in_0 = *x[0];
    auto &in_1 = *x[1];
    int64_t length_0 = in_0.numel();
    int64_t length_1 = in_1.numel();
    if (length_0 && length_1 && in_0.initialized() && in_1.initialized()) {
      auto result = EigenVector<T>::Flatten(*out);
      auto &place = *dev_ctx.eigen_device();
      auto in_0_e = EigenVector<T>::Flatten(in_0);
      auto in_1_e = EigenVector<T>::Flatten(in_1);
      result.device(place) = in_0_e + in_1_e;
    } else if (length_0 && in_0.initialized()) {
      auto result = EigenVector<T>::Flatten(*out);
      auto &place = *dev_ctx.eigen_device();
      result.device(place) = EigenVector<T>::Flatten(in_0);
    } else if (length_1 && in_1.initialized()) {
      auto result = EigenVector<T>::Flatten(*out);
      auto &place = *dev_ctx.eigen_device();
      result.device(place) = EigenVector<T>::Flatten(in_1);
    }
    return;
  }

  int start = in_place ? 1 : 0;
  if (!in_place) {
    funcs::SetConstant<Context, T> constant_functor;
    constant_functor(dev_ctx, out, static_cast<T>(0));
  }

  std::vector<const T *> in_data;
  int64_t lod_length = 0;
  bool dst_write = false;
  for (int i = start; i < in_num; ++i) {
    auto &in_i = *x[i];
    lod_length = in_i.numel();
    if (lod_length && in_i.initialized()) {
      in_data.emplace_back(in_i.data<T>());
    }
  }

  // if indata not null, merge into one kernel call.
  if (!in_data.empty()) {
    auto tmp_in_array =
        paddle::memory::Alloc(dev_ctx, in_data.size() * sizeof(T *));

    paddle::memory::Copy(dev_ctx.GetPlace(),
                         tmp_in_array->ptr(),
                         phi::CPUPlace(),
                         reinterpret_cast<void *>(in_data.data()),
                         in_data.size() * sizeof(T *),
                         dev_ctx.stream());

    T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());
    ComputeKernelParameter(lod_length);
    SumArrayCUDAKernel<T><<<grids, blocks, 0, stream>>>(in_array_data,
                                                        out->data<T>(),
                                                        lod_length,
                                                        in_data.size(),
                                                        dst_write | in_place);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(add_n,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddNKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {}
