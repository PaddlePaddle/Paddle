#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/add_n_kernel.h"

#include "paddle/phi/kernels/impl/add_n_kernel_impl.h"

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"

namespace phi {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

template <class T>
__global__ void Sum2CUDAKernel(const T *in_0,
                               const T *in_1,
                               T *out,
                               int64_t N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    out[id] = in_0[id] + in_1[id];
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void SumArrayCUDAKernel(
    T **in, T *out, int64_t N, size_t in_size, bool read_dst) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    T total(read_dst ? out[id] : static_cast<T>(0));
    for (int i = 0; i < in_size; ++i) {
      const T *tmp = in[i];
      if (tmp) {
        total += tmp[id];
      }
    }
    out[id] = total;
    id += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void SumSelectedRowsCUDAKernel(T **sr_in_out,
                                          int64_t N,
                                          size_t rows) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < N) {
    for (int i = 0; i < 2 * rows; i += 2) {
      const T *tmp = sr_in_out[i];
      T *tmp_out = sr_in_out[i + 1];
      if (tmp && tmp_out) {
        tmp_out[id] += tmp[id];
      }
    }
    id += blockDim.x * gridDim.x;
  }
}

template <typename T, typename Context>
void AddNKernel(const Context &dev_ctx,
                const std::vector<const TensorBase *> &x,
                DenseTensor *out) {
  const size_t in_num = x.size();

  constexpr size_t theory_sm_threads = 1024;
  auto stream = dev_ctx.stream();

  auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  auto sm_count = max_threads / theory_sm_threads;
  size_t tile_size = 0;
  dim3 grids;
  dim3 blocks;

  auto ComputeKernelParameter = [&](size_t length) {
    if (length >= max_threads)
      tile_size = 1024;
    else if (length < max_threads && length > sm_count * 128)
      tile_size = 512;
    else if (length <= sm_count * 128)
      tile_size = 256;
    grids = dim3(CEIL_DIV(length, tile_size), 1, 1);
    blocks = dim3(tile_size, 1, 1);
  };
  auto *out_ptr = dev_ctx.template Alloc<T>(out);
  bool in_place = false;
  if (x.size() > 0 && x[0]->initialized() && DenseTensor::classof(x[0])) {
    if ((static_cast<const DenseTensor *>(x[0]))->data() == out->data()) {
      in_place = true;
    }
  }

  if (!in_place && in_num >= 1 && DenseTensor::classof(x[0])) {
    auto &in_0_tensor = *(static_cast<const DenseTensor *>(x[0]));
    if (in_0_tensor.numel() > 0) {
      in_place = (in_0_tensor.data<T>() == out_ptr);
    }
  }

  // Sum of two tensors
  if (in_num == 2 && DenseTensor::classof(x[0]) && DenseTensor::classof(x[1])) {
    auto &in_0 = *(static_cast<const DenseTensor *>(x[0]));
    auto &in_1 = *(static_cast<const DenseTensor *>(x[1]));
    int64_t length_0 = in_0.numel();
    int64_t length_1 = in_1.numel();
    if (length_0 && length_1 && in_0.IsInitialized() && in_1.IsInitialized()) {
      auto result = EigenVector<T>::Flatten(*out);
      auto &place = *dev_ctx.eigen_device();
      auto in_0_e = EigenVector<T>::Flatten(in_0);
      auto in_1_e = EigenVector<T>::Flatten(in_1);
      result.device(place) = in_0_e + in_1_e;
    } else if (length_0 && in_0.IsInitialized()) {
      auto result = EigenVector<T>::Flatten(*out);
      auto &place = *dev_ctx.eigen_device();
      result.device(place) = EigenVector<T>::Flatten(in_0);
    } else if (length_1 && in_1.IsInitialized()) {
      auto result = EigenVector<T>::Flatten(*out);
      auto &place = *dev_ctx.eigen_device();
      result.device(place) = EigenVector<T>::Flatten(in_1);
    }
    return;
  }

  int start = in_place ? 1 : 0;
  if (!in_place) {
    phi::funcs::SetConstant<phi::GPUContext, T> constant_functor;
    constant_functor(dev_ctx, out, static_cast<T>(0));
  }

  std::vector<const T *> in_data;
  std::vector<int> selectrow_index;
  int64_t lod_length = 0;
  bool dst_write = false;
  for (int i = start; i < in_num; ++i) {
    if (DenseTensor::classof(x[i])) {
      auto &in_i = *(static_cast<const DenseTensor *>(x[i]));
      lod_length = in_i.numel();
      if (lod_length && in_i.IsInitialized()) {
        in_data.emplace_back(in_i.data<T>());
      }
    } else if (SelectedRows::classof(x[i])) {
      selectrow_index.push_back(i);
    }
  }

  // compute select rows separately.
  if (!selectrow_index.empty()) {
    std::vector<const T *> sr_in_out_data;
    size_t rows = 0;
    int64_t length = 0;
    for (auto index : selectrow_index) {
      auto &sr = *(static_cast<const SelectedRows *>(x[index]));
      auto &sr_value = sr.value();
      auto &sr_rows = sr.rows();

      auto row_numel = sr_value.numel() / sr_rows.size();
      auto out_dims = out->dims();

      PADDLE_ENFORCE_EQ(sr.height(),
                        out_dims[0],
                        errors::InvalidArgument(
                            "The table height of input must be same as output, "
                            "but received input height is %d"
                            ", output height is %d",
                            sr.height(),
                            out_dims[0]));
      PADDLE_ENFORCE_EQ(row_numel,
                        out->numel() / sr.height(),
                        errors::InvalidArgument(
                            "The table width of input must be same as output, "
                            "but received input width is %d"
                            ", output width is %d",
                            row_numel,
                            out->numel() / sr.height()));

      auto *sr_data = sr_value.data<T>();
      auto *sr_out_data = out->data<T>();
      rows += sr_rows.size();
      length = row_numel;

      for (size_t i = 0; i < sr_rows.size(); ++i) {
        sr_in_out_data.emplace_back(&sr_data[i * row_numel]);
        sr_in_out_data.emplace_back(&sr_out_data[sr_rows[i] * row_numel]);
      }
    }
    if (!sr_in_out_data.empty()) {
      auto tmp_sr_in_out_array = paddle::memory::Alloc(
          dev_ctx.GetPlace(), sr_in_out_data.size() * sizeof(T *));

      paddle::memory::Copy(dev_ctx.GetPlace(),
                           tmp_sr_in_out_array->ptr(),
                           phi::CPUPlace(),
                           reinterpret_cast<void *>(sr_in_out_data.data()),
                           sr_in_out_data.size() * sizeof(T *),
                           dev_ctx.stream());

      T **sr_in_out_array_data =
          reinterpret_cast<T **>(tmp_sr_in_out_array->ptr());

      ComputeKernelParameter(length);
      SumSelectedRowsCUDAKernel<T>
          <<<grids, blocks, 0, stream>>>(sr_in_out_array_data, length, rows);
      dst_write = true;
    }
  }
  // if indata not null, merge into one kernel call.
  if (!in_data.empty()) {
    auto tmp_in_array =
        paddle::memory::Alloc(dev_ctx.GetPlace(), in_data.size() * sizeof(T *));

    paddle::memory::Copy(dev_ctx.GetPlace(),
                         tmp_in_array->ptr(),
                         phi::CPUPlace(),
                         reinterpret_cast<void *>(in_data.data()),
                         in_data.size() * sizeof(T *),
                         dev_ctx.stream());

    T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());
    ComputeKernelParameter(lod_length);
    SumArrayCUDAKernel<T><<<grids, blocks, 0, stream>>>(in_array_data,
                                                        out->data<T>(),
                                                        lod_length,
                                                        in_data.size(),
                                                        dst_write | in_place);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(add_n,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddNKernel,
                   float,
                   double,
                   int,
                   phi::dtype::bfloat16,
                   phi::dtype::float16,
                   int64_t) {}

PD_REGISTER_KERNEL(add_n_array,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddNArrayKernel,
                   float,
                   double,
                   int,
                   phi::dtype::bfloat16,
                   phi::dtype::float16,
                   int64_t) {}
