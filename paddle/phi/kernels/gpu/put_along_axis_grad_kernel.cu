// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/put_along_axis_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/place.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/core/utils/data_type.h"
#include "paddle/phi/kernels/funcs/gather_scatter_functor.h"

namespace phi {

template <typename T, typename Context>
void PutAlongAxisGradKernel(const Context& dev_ctx,
                            const DenseTensor& x,
                            const DenseTensor& index,
                            const DenseTensor& value,
                            const DenseTensor& out,
                            const DenseTensor& out_grad,
                            int axis,
                            const std::string& reduce,
                            bool include_self,
                            DenseTensor* x_grad,
                            DenseTensor* value_grad) {
  PADDLE_ENFORCE_EQ(dev_ctx.GetPlace().GetType() == phi::AllocationType::GPU,
                    true,
                    errors::PreconditionNotMet(
                        "PutAlongAxisGradOpCUDAKernel only runs on GPU."));

  const auto& index_type = index.dtype();
  if (x_grad) {
    phi::Copy(dev_ctx, out_grad, dev_ctx.GetPlace(), false, x_grad);
    if (!include_self || reduce == "assign") {
      if (index_type == DataType::INT32) {
        phi::funcs::gpu_scatter_input_grad_kernel<T, int32_t>(
            out_grad, axis, index, *x_grad, include_self, dev_ctx);
      } else {
        phi::funcs::gpu_scatter_input_grad_kernel<T, int64_t>(
            out_grad, axis, index, *x_grad, include_self, dev_ctx);
      }
    } else if (reduce == "multiply" || reduce == "mul" || reduce == "amin" ||
               reduce == "amax") {
      if (index_type == DataType::INT32) {
        phi::funcs::gpu_scatter_mul_min_max_input_grad_kernel<T, int32_t>(
            out_grad,
            axis,
            index,
            out,
            x,
            value,
            *x_grad,
            reduce,
            include_self,
            dev_ctx);
      } else {
        phi::funcs::gpu_scatter_mul_min_max_input_grad_kernel<T, int64_t>(
            out_grad,
            axis,
            index,
            out,
            x,
            value,
            *x_grad,
            reduce,
            include_self,
            dev_ctx);
      }
    } else if (reduce == "mean") {
      if (index_type == DataType::INT32) {
        phi::funcs::gpu_scatter_mean_input_grad_kernel<T, int32_t>(
            out_grad, axis, index, *x_grad, include_self, dev_ctx);
      } else {
        phi::funcs::gpu_scatter_mean_input_grad_kernel<T, int64_t>(
            out_grad, axis, index, *x_grad, include_self, dev_ctx);
      }
    }
  }
  if (value_grad) {
    value_grad->Resize(index.dims());
    dev_ctx.template Alloc<T>(value_grad);
    auto* grad_data = value_grad->data<T>();
    int64_t grad_size = value_grad->numel();
    hipMemset(grad_data, 0, sizeof(T) * grad_size);
    if (reduce == "assign") {
      if (index_type == DataType::INT32) {
        phi::funcs::gpu_scatter_value_grad_kernel<T, int32_t>(
            out_grad, axis, index, *value_grad, include_self, dev_ctx);
      } else if (index_type == DataType::INT64) {
        phi::funcs::gpu_scatter_value_grad_kernel<T, int64_t>(
            out_grad, axis, index, *value_grad, include_self, dev_ctx);
      }
    } else if (reduce == "add" || reduce == "mean") {
      if (index_type == DataType::INT32) {
        phi::funcs::gpu_scatter_add_mean_value_grad_kernel<T, int32_t>(
            out_grad,
            axis,
            index,
            out,
            x,
            value,
            *value_grad,
            reduce,
            include_self,
            dev_ctx);
      } else {
        phi::funcs::gpu_scatter_add_mean_value_grad_kernel<T, int64_t>(
            out_grad,
            axis,
            index,
            out,
            x,
            value,
            *value_grad,
            reduce,
            include_self,
            dev_ctx);
      }
    } else if (reduce == "mul" || reduce == "multiply" || reduce == "amin" ||
               reduce == "amax") {
      if (index_type == DataType::INT32) {
        phi::funcs::gpu_scatter_mul_min_max_value_grad_kernel<T, int32_t>(
            out_grad,
            axis,
            index,
            out,
            x,
            value,
            *value_grad,
            reduce,
            include_self,
            dev_ctx);
      } else {
        phi::funcs::gpu_scatter_mul_min_max_value_grad_kernel<T, int64_t>(
            out_grad,
            axis,
            index,
            out,
            x,
            value,
            *value_grad,
            reduce,
            include_self,
            dev_ctx);
      }
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(put_along_axis_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::PutAlongAxisGradKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
