/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/activation_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/impl/activation_grad_impl.h"

namespace phi {

template <typename T, typename Context, typename Functor>
void ActivationGradGPUImpl(const Context& dev_ctx,
                           const DenseTensor* x,
                           const DenseTensor* out,
                           const DenseTensor* d_out,
                           DenseTensor* d_x,
                           const Functor& functor) {
  if (static_cast<int>(Functor::FwdDeps()) &
      static_cast<int>(funcs::ActBwdOpFwdDeps::kDepOut)) {
    PADDLE_ENFORCE_NOT_NULL(
        out, errors::NotFound("The input DenseTensor Out can not be nullptr"));
  }
  PADDLE_ENFORCE_NOT_NULL(
      d_out, errors::NotFound("The input DenseTensor dOut can not be nullptr"));
  PADDLE_ENFORCE_NOT_NULL(
      d_x, errors::NotFound("The output DenseTensor dX can not be nullptr"));
  if (!out) {
    out = d_out;  // fake out
  }
  if (static_cast<int>(Functor::FwdDeps()) &
      static_cast<int>(funcs::ActBwdOpFwdDeps::kDepX)) {
    PADDLE_ENFORCE_NOT_NULL(
        x, errors::NotFound("The input DenseTensor X can not be nullptr"));
  } else {
    VLOG(10) << "Inplace activation of Op Functor: " << typeid(Functor).name();
    x = d_x;
  }

  dev_ctx.template Alloc<T>(d_x);

  std::vector<const DenseTensor*> ins = {d_out};
  std::vector<DenseTensor*> outs = {d_x};

  if (static_cast<int>(Functor::FwdDeps()) ==
      static_cast<int>(funcs::ActBwdOpFwdDeps::kDepOut)) {
    // Only need forward output Out
    ins.push_back(out);
    funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
  } else if (static_cast<int>(Functor::FwdDeps()) ==
             static_cast<int>(funcs::ActBwdOpFwdDeps::kDepX)) {
    // Only need forward input X
    ins.push_back(x);
    funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
  } else {
    funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
  }
}

#define DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(name, functor_class) \
  template <typename T, typename Context>                           \
  void name##GradKernel(const Context& dev_ctx,                     \
                        const DenseTensor& x,                       \
                        const DenseTensor& dout,                    \
                        DenseTensor* dx) {                          \
    funcs::functor_class<T> functor;                                \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>(     \
        dev_ctx, &x, nullptr, &dout, dx, functor);                  \
  }

#define DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(         \
    name, functor_class, attr)                                  \
  template <typename T, typename Context>                       \
  void name##GradKernel(const Context& dev_ctx,                 \
                        const DenseTensor& x,                   \
                        const DenseTensor& dout,                \
                        float attr,                             \
                        DenseTensor* dx) {                      \
    funcs::functor_class<T> functor;                            \
    auto attrs = functor.GetAttrs();                            \
    *(attrs[0].second) = attr;                                  \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>( \
        dev_ctx, &x, nullptr, &dout, dx, functor);              \
  }

#define DEFINE_GPU_ACT_GRAD_KERNEL_WITH_TWO_ATTRS_DEPX(         \
    name, functor_class, attr1, attr2)                          \
  template <typename T, typename Context>                       \
  void name##GradKernel(const Context& dev_ctx,                 \
                        const DenseTensor& x,                   \
                        const DenseTensor& dout,                \
                        float attr1,                            \
                        float attr2,                            \
                        DenseTensor* dx) {                      \
    funcs::functor_class<T> functor;                            \
    auto attrs = functor.GetAttrs();                            \
    *(attrs[0].second) = attr1;                                 \
    *(attrs[1].second) = attr2;                                 \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>( \
        dev_ctx, &x, nullptr, &dout, dx, functor);              \
  }

#define DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(name, functor_class) \
  template <typename T, typename Context>                             \
  void name##GradKernel(const Context& dev_ctx,                       \
                        const DenseTensor& out,                       \
                        const DenseTensor& dout,                      \
                        DenseTensor* dx) {                            \
    funcs::functor_class<T> functor;                                  \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>(       \
        dev_ctx, nullptr, &out, &dout, dx, functor);                  \
  }

#define DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPOUT(       \
    name, functor_class, attr)                                  \
  template <typename T, typename Context>                       \
  void name##GradKernel(const Context& dev_ctx,                 \
                        const DenseTensor& out,                 \
                        const DenseTensor& dout,                \
                        float attr,                             \
                        DenseTensor* dx) {                      \
    funcs::functor_class<T> functor;                            \
    auto attrs = functor.GetAttrs();                            \
    *(attrs[0].second) = attr;                                  \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>( \
        dev_ctx, nullptr, &out, &dout, dx, functor);            \
  }

#define DEFINE_GPU_ACT_GRAD_KERNEL_WITH_TWO_ATTRS_DEPOUT(       \
    name, functor_class, attr1, attr2)                          \
  template <typename T, typename Context>                       \
  void name##GradKernel(const Context& dev_ctx,                 \
                        const DenseTensor& out,                 \
                        const DenseTensor& dout,                \
                        float attr1,                            \
                        float attr2,                            \
                        DenseTensor* dx) {                      \
    funcs::functor_class<T> functor;                            \
    auto attrs = functor.GetAttrs();                            \
    *(attrs[0].second) = attr1;                                 \
    *(attrs[1].second) = attr2;                                 \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>( \
        dev_ctx, nullptr, &out, &dout, dx, functor);            \
  }

#define DEFINE_GPU_ACTIVATION_GRAD_KERNEL_NODEP(name, functor_class)      \
  template <typename T, typename Context>                                 \
  void name##GradKernel(                                                  \
      const Context& dev_ctx, const DenseTensor& dout, DenseTensor* dx) { \
    funcs::functor_class<T> functor;                                      \
    ActivationGradGPUImpl<T, Context, funcs::functor_class<T>>(           \
        dev_ctx, nullptr, nullptr, &dout, dx, functor);                   \
  }

DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Relu, CudaReluGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Tanh, CudaTanhGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Sigmoid, CudaSigmoidGradFunctor);

DEFINE_GPU_ACTIVATION_GRAD_KERNEL_NODEP(Round, CudaZeroGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_NODEP(Floor, CudaZeroGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_NODEP(Ceil, CudaZeroGradFunctor);

DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Cos, CudaCosGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Tan, CudaTanGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Acos, CudaAcosGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Sin, CudaSinGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Asin, CudaAsinGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Atan, CudaAtanGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Sinh, CudaSinhGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Cosh, CudaCoshGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Asinh, CudaAsinhGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Acosh, CudaAcoshGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Atanh, CudaAtanhGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(TanhShrink, CudaTanhShrinkGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Silu, CudaSiluGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Square, CudaSquareGradFunctor);

DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Exp, CudaExpGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Expm1, CudaExpm1GradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Reciprocal, CudaReciprocalGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Sqrt, CudaSqrtGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPOUT(Rsqrt, CudaRsqrtGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Softsign, CudaSoftsignGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(LogSigmoid, CudaLogSigmoidGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Log, CudaLogGradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Log2, CudaLog2GradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Log10, CudaLog10GradFunctor);
DEFINE_GPU_ACTIVATION_GRAD_KERNEL_DEPX(Log1p, CudaLog1pGradFunctor);

DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(LeakyRelu,
                                               CudaLeakyReluGradFunctor,
                                               alpha);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(ThresholdedRelu,
                                               CudaThresholdedReluGradFunctor,
                                               threshold);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(SoftShrink,
                                               CudaSoftShrinkGradFunctor,
                                               lambda);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(HardShrink,
                                               CudaHardShrinkGradFunctor,
                                               threshold);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(Swish,
                                               CudaSwishGradFunctor,
                                               beta);

DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(Mish,
                                               CudaMishGradFunctor,
                                               threshold);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPX(Celu,
                                               CudaCELUGradFunctor,
                                               alpha);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_ONE_ATTRS_DEPOUT(Relu6,
                                                 CudaRelu6GradFunctor,
                                                 threshold);

DEFINE_GPU_ACT_GRAD_KERNEL_WITH_TWO_ATTRS_DEPX(BRelu,
                                               CudaBReluGradFunctor,
                                               t_min,
                                               t_max);

DEFINE_GPU_ACT_GRAD_KERNEL_WITH_TWO_ATTRS_DEPX(STanh,
                                               CudaSTanhGradFunctor,
                                               scale_a,
                                               scale_b);

DEFINE_GPU_ACT_GRAD_KERNEL_WITH_TWO_ATTRS_DEPX(Softplus,
                                               CudaSoftplusGradFunctor,
                                               beta,
                                               threshold);
DEFINE_GPU_ACT_GRAD_KERNEL_WITH_TWO_ATTRS_DEPOUT(HardSigmoid,
                                                 CudaHardSigmoidGradFunctor,
                                                 slope,
                                                 offset);

template <typename T, typename Context>
void EluGradKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& out,
                   const DenseTensor& dout,
                   float alpha,
                   DenseTensor* dx) {
  dev_ctx.template Alloc<T>(dx);
  std::vector<const DenseTensor*> ins = {&dout, &out};
  std::vector<DenseTensor*> outs = {dx};
  if (alpha > 0) {
    funcs::CudaELUGradFunctor<T> functor;
    functor.alpha = alpha;
    funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
  } else {
    funcs::CudaELUGradNegativeAlphaFunctor<T> functor;
    functor.alpha = alpha;
    ins.push_back(&x);
    funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
  }
}

template <typename T, typename Context>
void HardSwishGradKernel(const Context& dev_ctx,
                         const DenseTensor& x,
                         const DenseTensor& dout,
                         float threshold,
                         float scale,
                         float offset,
                         DenseTensor* dx) {
  funcs::CudaHardSwishGradFunctor<T> functor;
  auto attrs = functor.GetAttrs();
  *(attrs[0].second) = threshold;
  *(attrs[1].second) = scale;
  *(attrs[2].second) = offset;
  ActivationGradGPUImpl<T, Context, funcs::CudaHardSwishGradFunctor<T>>(
      dev_ctx, &x, nullptr, &dout, dx, functor);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(relu_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReluGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(relu_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReluDoubleGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(relu_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReluGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(relu_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReluDoubleGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#endif

#define PD_REGISTER_ACTIVATION_GRAD_KERNEL(name, func) \
  PD_REGISTER_KERNEL(name,                             \
                     GPU,                              \
                     ALL_LAYOUT,                       \
                     phi::func,                        \
                     float,                            \
                     double,                           \
                     phi::dtype::float16,              \
                     phi::dtype::bfloat16) {}

PD_REGISTER_ACTIVATION_GRAD_KERNEL(sin_grad, SinGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(cos_grad, CosGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(tan_grad, TanGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(acos_grad, AcosGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(asin_grad, AsinGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(atan_grad, AtanGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(sinh_grad, SinhGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(cosh_grad, CoshGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(asinh_grad, AsinhGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(acosh_grad, AcoshGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(atanh_grad, AtanhGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(tanh_grad, TanhGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(tanh_double_grad, TanhDoubleGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(tanh_triple_grad, TanhTripleGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(brelu_grad, BReluGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(leaky_relu_grad, LeakyReluGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(leaky_relu_double_grad,
                                   LeakyReluDoubleGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(thresholded_relu_grad,
                                   ThresholdedReluGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(relu6_grad, Relu6GradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(mish_grad, MishGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(stanh_grad, STanhGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(reciprocal_grad, ReciprocalGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(softplus_grad, SoftplusGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(sqrt_grad, SqrtGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(sqrt_double_grad, SqrtDoubleGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(rsqrt_grad, RsqrtGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(rsqrt_double_grad, RsqrtDoubleGradKernel)

PD_REGISTER_KERNEL(exp_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ExpGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}

PD_REGISTER_ACTIVATION_GRAD_KERNEL(soft_shrink_grad, SoftShrinkGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(hard_shrink_grad, HardShrinkGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(tanh_shrink_grad, TanhShrinkGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(silu_grad, SiluGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(elu_grad, EluGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(elu_double_grad, EluDoubleGradKernel)

PD_REGISTER_KERNEL(expm1_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::Expm1GradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(logit_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::LogitGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(square_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SquareGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(square_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SquareDoubleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_ACTIVATION_GRAD_KERNEL(softsign_grad, SoftsignGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(sigmoid_grad, SigmoidGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(sigmoid_double_grad, SigmoidDoubleGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(sigmoid_triple_grad, SigmoidTripleGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(hard_sigmoid_grad, HardSigmoidGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(logsigmoid_grad, LogSigmoidGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(log_grad, LogGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(log2_grad, Log2GradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(log10_grad, Log10GradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(log1p_grad, Log1pGradKernel)
PD_REGISTER_KERNEL(log_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::LogDoubleGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_ACTIVATION_GRAD_KERNEL(hard_swish_grad, HardSwishGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(swish_grad, SwishGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(round_grad, RoundGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(floor_grad, FloorGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(ceil_grad, CeilGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(celu_grad, CeluGradKernel)
PD_REGISTER_ACTIVATION_GRAD_KERNEL(celu_double_grad, CeluDoubleGradKernel)

PD_REGISTER_KERNEL(pow_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::PowGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
