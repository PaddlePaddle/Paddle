#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/tril_indices_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

  __device__
  inline int64_t resolve_root_int(int64_t b, 
                                  int64_t cX4, 
                                  int64_t x, 
                                  int32_t sign) {
  int64_t bXb_cX4 = b*b - cX4;
  // potential precision loss could occur here when casting int64_t (63 bits
  // precision) to double (52 bits precision)
  double sr = ::sqrt((double)bXb_cX4);
  int64_t res = ::__double2ll_rd((-b + sign * sr)/2);

  // have to cast double to int64_t, otherwise it would only compare up to the
  // precision of a double variable, ignoring the precision loss
  if (bXb_cX4 != (int64_t) (sr * sr)) {
    // handle precision loss by using binary search
    int64_t llsr = ::__double2ll_rd(sr);
    // Use the following math to reduce search space.
    // Suppose z is the accurate result of sqrt(bXb_cX4) without precision loss
    // let d = abs(bXb_cX4 - llsr * llsr), then we have:
    // z = sqrt(bXb_cX4) <= sqrt(llsr * llsr + d) <= llsr + sqrt(d)
    // z = sqrt(bXb_cX4) >= sqrt(llsr * llsr - d) >= llsr - sqrt(d)
    // Hence, it is sufficient to search range [llsr - sqrt(d), llsr + sqrt(d)).
    // And the true value of row would also be with in range,
    //            [res - sqrt(d), res + sqrt(d) + 1)
    // as the denominator would only reduce the precision penalty.
    int64_t diff =
      ::__double2ll_ru(::sqrt(::fabs((double)(bXb_cX4 - llsr * llsr))));
    // l never exceeds (could equal to) the target row index
    auto l = res > diff ? res - diff : 0;
    // r is always larger than the target row index
    auto r = res + diff + 1;

    // binary search for the correct answer
    x <<= 1; // the loop always compares with 2x, so do it once here
    while (l + 1 < r) {
      auto m = (l + r) >> 1;
      // for tril:
      //    b = 2f - 1, sign = 1, hence (2f + m - 1) * m / 2
      // for triu:
      //    b = -2f - 1, sign = -1, hence (2f - m + 1) * m / 2
      if (sign * (b + m) * m > x) {
        r = m;
      } else {
        l = m;
      }
    }
    res = l;
  }

  return res;
}

  __device__
  inline void get_coordinate_in_tril_trapezoid(int64_t f, 
                                               int64_t x,
                                               int64_t & row, 
                                               int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = f - 1;
  auto cX4 = - (x << 3); // 4 * c = 4 * (-2x) = -8x;
  row = resolve_root_int(b, cX4, x, 1);
  col = x - ((f + row - 1) * row >> 1);
}
  void tril_indices_kernel(DenseTensor* out_data,
                          int64_t row_offset,
                          int64_t m_first_row,
                          int64_t col,
                          int64_t trapezoid_size,
                          int64_t tril_size){
  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (linear_index < tril_size) {
    int64_t r, c;
    if (linear_index < trapezoid_size) {
      // the coordinate is within the top trapezoid
      get_coordinate_in_tril_trapezoid(m_first_row, linear_index, r, c);
    } else {
      // the coordinate falls in the bottom rectangle
      auto surplus = linear_index - trapezoid_size;
      // add the height of trapezoid: m_last_row (col) - m_first_row + 1
      r = surplus / col + col - m_first_row + 1;
      c = surplus % col;
    }
    r += row_offset;

    out_data[linear_index] = r;
    out_data[linear_index + tril_size] = c;
  }
}


template <typename T, typename Context>
void TrilIndicesKernel(const Context& dev_ctx,
                      int rows,
                      int cols,
                      int offset,
                      DataType dtype,
                      DenseTensor* out) {
  T* out_data = dev_ctx.template Alloc<T>(out);
  auto out_dims = out->dims();
  int64_t tril_size = out_dims[1];

  if (tril_size > 0) {
    auto m_first_row = offset > 0 ? std::min<int64_t>(col, 1 + offset) : row + offset > 0; 
    auto trapezoid_row_offset = std::max<int64_t>(0, -offset);
    auto rectangle_row_offset = trapezoid_row_offset + col - m_first_row + 1;
    int64_t rectangle_size = 0;
    if (rectangle_row_offset < row) {
        rectangle_size = (row - rectangle_row_offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;
    // using tril_size instead of out_data.numel(), as each thread takes care of
    // two elements in the out_data.
   
    cuda::getApplyGrid(tril_size, dim_grid, out_data.get_device());//"unable to get dim grid";

    tril_indices_kernel<<<dim_grid, dim_block, 0, cuda::getCurrentCUDAStream()>>>(
        out_data.data_ptr<scalar_t>(),
        trapezoid_row_offset,
        m_first_row,
        col,
        tril_size - rectangle_size,
        tril_size);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    
  }

  return out_data;
}

}  // namespace phi

PD_REGISTER_KERNEL(tril_indices,
                   GPU,
                   ALL_LAYOUT,
                   phi::TrilIndicesKernel,
                   int,
                   int64_t,
                   ) {}
