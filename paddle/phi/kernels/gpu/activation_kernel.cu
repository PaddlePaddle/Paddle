#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/activation_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/impl/activation_grad_impl.h"
#include "paddle/phi/kernels/impl/activation_impl.h"

namespace phi {

template <typename T, typename Context, typename Functor>
void ActivationGPUImpl(const Context& dev_ctx,
                       const DenseTensor& x,
                       DenseTensor* out,
                       const Functor& functor) {
  PADDLE_ENFORCE_NOT_NULL(out,
                          errors::NotFound("Output Out should not be nullptr"));
  dev_ctx.template Alloc<T>(out);
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
}

#define DEFINE_GPU_ACTIVATION_KERNEL(name, functor_class)               \
  template <typename T, typename Context>                               \
  void name##Kernel(                                                    \
      const Context& dev_ctx, const DenseTensor& x, DenseTensor* out) { \
    funcs::functor_class<T> functor;                                    \
    ActivationGPUImpl<T, Context, funcs::functor_class<T>>(             \
        dev_ctx, x, out, functor);                                      \
  }

#define DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(name, functor_class, attr) \
  template <typename T, typename Context>                               \
  void name##Kernel(const Context& dev_ctx,                             \
                    const DenseTensor& x,                               \
                    float attr,                                         \
                    DenseTensor* out) {                                 \
    funcs::functor_class<T> functor;                                    \
    auto attrs = functor.GetAttrs();                                    \
    *(attrs[0].second) = attr;                                          \
    ActivationGPUImpl<T, Context, funcs::functor_class<T>>(             \
        dev_ctx, x, out, functor);                                      \
  }

#define DEFINE_GPU_ACT_KERNEL_WITH_TWO_ATTRS(               \
    name, functor_class, attr1, attr2)                      \
  template <typename T, typename Context>                   \
  void name##Kernel(const Context& dev_ctx,                 \
                    const DenseTensor& x,                   \
                    float attr1,                            \
                    float attr2,                            \
                    DenseTensor* out) {                     \
    funcs::functor_class<T> functor;                        \
    auto attrs = functor.GetAttrs();                        \
    *(attrs[0].second) = attr1;                             \
    *(attrs[1].second) = attr2;                             \
    ActivationGPUImpl<T, Context, funcs::functor_class<T>>( \
        dev_ctx, x, out, functor);                          \
  }

DEFINE_GPU_ACTIVATION_KERNEL(Cos, CudaCosFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Tan, CudaTanFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Acos, CudaAcosFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Sin, CudaSinFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Asin, CudaAsinFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Atan, CudaAtanFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Sinh, CudaSinhFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Cosh, CudaCoshFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Asinh, CudaAsinhFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Acosh, CudaAcoshFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Atanh, CudaAtanhFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Relu, CudaReluFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Tanh, CudaTanhFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(TanhShrink, CudaTanhShrinkFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Silu, CudaSiluFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Exp, CudaExpFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Expm1, CudaExpm1Functor)
DEFINE_GPU_ACTIVATION_KERNEL(Reciprocal, CudaReciprocalFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Square, CudaSquareFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Sqrt, CudaSqrtFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Rsqrt, CudaRsqrtFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Softsign, CudaSoftsignFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Sigmoid, CudaSigmoidFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(LogSigmoid, CudaLogSigmoidFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Log, CudaLogFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Log2, CudaLog2Functor)
DEFINE_GPU_ACTIVATION_KERNEL(Log10, CudaLog10Functor)
DEFINE_GPU_ACTIVATION_KERNEL(Log1p, CudaLog1pFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Round, CudaRoundFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Floor, CudaFloorFunctor)
DEFINE_GPU_ACTIVATION_KERNEL(Ceil, CudaCeilFunctor)

DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(LeakyRelu, CudaLeakyReluFunctor, alpha)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(ThresholdedRelu,
                                     CudaThresholdedReluFunctor,
                                     threshold)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(Relu6, CudaRelu6Functor, threshold)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(HardShrink,
                                     CudaHardShrinkFunctor,
                                     threshold)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(SoftShrink, CudaSoftShrinkFunctor, lambda)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(Elu, CudaELUFunctor, alpha)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(Swish, CudaSwishFunctor, beta)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(Mish, CudaMishFunctor, threshold)
DEFINE_GPU_ACT_KERNEL_WITH_ONE_ATTRS(Celu, CudaCELUFunctor, alpha)

DEFINE_GPU_ACT_KERNEL_WITH_TWO_ATTRS(BRelu, CudaBReluFunctor, t_min, t_max)
DEFINE_GPU_ACT_KERNEL_WITH_TWO_ATTRS(Stanh, CudaSTanhFunctor, scale_a, scale_b)
DEFINE_GPU_ACT_KERNEL_WITH_TWO_ATTRS(Softplus,
                                     CudaSoftplusFunctor,
                                     beta,
                                     threshold)
DEFINE_GPU_ACT_KERNEL_WITH_TWO_ATTRS(HardSigmoid,
                                     CudaHardSigmoidFunctor,
                                     slope,
                                     offset)
DEFINE_GPU_ACT_KERNEL_WITH_TWO_ATTRS(Selu, CudaSeluFunctor, scale, alpha)

template <typename T, typename Context>
void HardSwishKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     float threshold,
                     float scale,
                     float offset,
                     DenseTensor* out) {
  funcs::CudaHardSwishFunctor<T> functor;
  auto attrs = functor.GetAttrs();
  *(attrs[0].second) = threshold;
  *(attrs[1].second) = scale;
  *(attrs[2].second) = offset;
  ActivationGPUImpl<T, Context, funcs::CudaHardSwishFunctor<T>>(
      dev_ctx, x, out, functor);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(relu,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReluKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(relu,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReluKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#endif

#define PD_REGISTER_ACTIVATION_KERNEL(name, func) \
  PD_REGISTER_KERNEL(name,                        \
                     GPU,                         \
                     ALL_LAYOUT,                  \
                     phi::func,                   \
                     float,                       \
                     double,                      \
                     phi::dtype::float16,         \
                     phi::dtype::bfloat16) {}

PD_REGISTER_ACTIVATION_KERNEL(sin, SinKernel)
PD_REGISTER_ACTIVATION_KERNEL(cos, CosKernel)
PD_REGISTER_ACTIVATION_KERNEL(tan, TanKernel)
PD_REGISTER_ACTIVATION_KERNEL(acos, AcosKernel)
PD_REGISTER_ACTIVATION_KERNEL(asin, AsinKernel)
PD_REGISTER_ACTIVATION_KERNEL(atan, AtanKernel)
PD_REGISTER_ACTIVATION_KERNEL(sinh, SinhKernel)
PD_REGISTER_ACTIVATION_KERNEL(cosh, CoshKernel)
PD_REGISTER_ACTIVATION_KERNEL(asinh, AsinhKernel)
PD_REGISTER_ACTIVATION_KERNEL(acosh, AcoshKernel)
PD_REGISTER_ACTIVATION_KERNEL(atanh, AtanhKernel)
PD_REGISTER_ACTIVATION_KERNEL(tanh, TanhKernel)
PD_REGISTER_ACTIVATION_KERNEL(brelu, BReluKernel)
PD_REGISTER_ACTIVATION_KERNEL(thresholded_relu, ThresholdedReluKernel)
PD_REGISTER_ACTIVATION_KERNEL(relu6, Relu6Kernel)
PD_REGISTER_ACTIVATION_KERNEL(leaky_relu, LeakyReluKernel)
PD_REGISTER_ACTIVATION_KERNEL(mish, MishKernel)
PD_REGISTER_ACTIVATION_KERNEL(stanh, StanhKernel)
PD_REGISTER_ACTIVATION_KERNEL(reciprocal, ReciprocalKernel)
PD_REGISTER_ACTIVATION_KERNEL(sqrt, SqrtKernel)
PD_REGISTER_ACTIVATION_KERNEL(rsqrt, RsqrtKernel)
PD_REGISTER_ACTIVATION_KERNEL(softplus, SoftplusKernel)

PD_REGISTER_KERNEL(exp,
                   GPU,
                   ALL_LAYOUT,
                   phi::ExpKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(expm1,
                   GPU,
                   ALL_LAYOUT,
                   phi::Expm1Kernel,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(logit,
                   GPU,
                   ALL_LAYOUT,
                   phi::LogitKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(square,
                   GPU,
                   ALL_LAYOUT,
                   phi::SquareKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_ACTIVATION_KERNEL(hard_shrink, HardShrinkKernel)
PD_REGISTER_ACTIVATION_KERNEL(soft_shrink, SoftShrinkKernel)
PD_REGISTER_ACTIVATION_KERNEL(tanh_shrink, TanhShrinkKernel)
PD_REGISTER_ACTIVATION_KERNEL(elu, EluKernel)
PD_REGISTER_ACTIVATION_KERNEL(silu, SiluKernel)
PD_REGISTER_ACTIVATION_KERNEL(softsign, SoftsignKernel)
PD_REGISTER_ACTIVATION_KERNEL(sigmoid, SigmoidKernel)
PD_REGISTER_ACTIVATION_KERNEL(logsigmoid, LogSigmoidKernel)
PD_REGISTER_ACTIVATION_KERNEL(hard_sigmoid, HardSigmoidKernel)
PD_REGISTER_ACTIVATION_KERNEL(log, LogKernel)
PD_REGISTER_ACTIVATION_KERNEL(log2, Log2Kernel)
PD_REGISTER_ACTIVATION_KERNEL(log10, Log10Kernel)
PD_REGISTER_ACTIVATION_KERNEL(log1p, Log1pKernel)
PD_REGISTER_ACTIVATION_KERNEL(hard_swish, HardSwishKernel)
PD_REGISTER_ACTIVATION_KERNEL(swish, SwishKernel)
PD_REGISTER_ACTIVATION_KERNEL(round, RoundKernel)
PD_REGISTER_ACTIVATION_KERNEL(floor, FloorKernel)
PD_REGISTER_ACTIVATION_KERNEL(ceil, CeilKernel)
PD_REGISTER_ACTIVATION_KERNEL(celu, CeluKernel)
PD_REGISTER_KERNEL(pow,
                   GPU,
                   ALL_LAYOUT,
                   phi::PowKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
PD_REGISTER_KERNEL(selu,
                   GPU,
                   ALL_LAYOUT,
                   phi::SeluKernel,
                   float,
                   double,
                   phi::dtype::bfloat16) {}
