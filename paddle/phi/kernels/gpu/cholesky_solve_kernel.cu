// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifndef PADDLE_WITH_HIP
// HIP not support cusolver

#include "paddle/fluid/platform/enforce.h"
#include "paddle/phi/backends/dynload/cusolver.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/complex.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/lapack/lapack_function.h"
#include "paddle/phi/kernels/impl/cholesky_solve_kernel_impl.h"

namespace phi {

template <typename T>
void cusolver_potrs(const solverHandle_t &handle,
                    hipblasFillMode_t uplo,
                    int M,
                    int N,
                    T *Adata,
                    int lda,
                    T *Bdata,
                    int ldb,
                    int *devInfo);

template <>
void cusolver_potrs<float>(const solverHandle_t &handle,
                           hipblasFillMode_t uplo,
                           int M,
                           int N,
                           float *Adata,
                           int lda,
                           float *Bdata,
                           int ldb,
                           int *devInfo) {
  PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipsolverDnSpotrs(
      handle, uplo, M, N, Adata, lda, Bdata, ldb, devInfo));
}

template <>
void cusolver_potrs<double>(const solverHandle_t &handle,
                            hipblasFillMode_t uplo,
                            int M,
                            int N,
                            double *Adata,
                            int lda,
                            double *Bdata,
                            int ldb,
                            int *devInfo) {
  PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipsolverDnDpotrs(
      handle, uplo, M, N, Adata, lda, Bdata, ldb, devInfo));
}

template <>
void cusolver_potrs<phi::dtype::complex<float>>(
    const solverHandle_t &handle,
    hipblasFillMode_t uplo,
    int M,
    int N,
    phi::dtype::complex<float> *Adata,
    int lda,
    phi::dtype::complex<float> *Bdata,
    int ldb,
    int *devInfo) {
  PADDLE_ENFORCE_GPU_SUCCESS(
      dynload::hipsolverDnCpotrs(handle,
                                uplo,
                                M,
                                N,
                                reinterpret_cast<const hipComplex *>(Adata),
                                lda,
                                reinterpret_cast<hipComplex *>(Bdata),
                                ldb,
                                devInfo));
}

template <>
void cusolver_potrs<phi::dtype::complex<double>>(
    const hipsolverHandle_t &handle,
    hipblasFillMode_t uplo,
    int M,
    int N,
    phi::dtype::complex<double> *Adata,
    int lda,
    phi::dtype::complex<double> *Bdata,
    int ldb,
    int *devInfo) {
  PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipsolverDnZpotrs(
      handle,
      uplo,
      M,
      N,
      reinterpret_cast<const hipDoubleComplex *>(Adata),
      lda,
      reinterpret_cast<hipDoubleComplex *>(Bdata),
      ldb,
      devInfo));
}

template <typename T>
class CholeskySolveFunctor<T, GPUContext> {
 public:
  void operator()(const GPUContext &dev_ctx,
                  bool upper,
                  int M,
                  int N,
                  T *Adata,
                  int lda,
                  T *Bdata,
                  int *devInfo) {
    hipblasFillMode_t uplo =
        upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
    auto handle = dev_ctx.cusolver_dn_handle();
    cusolver_potrs<T>(handle, uplo, M, N, Adata, lda, Bdata, lda, devInfo);
  }
};

}  // namespace phi

PD_REGISTER_KERNEL(cholesky_solve,  // cuda_only
                   GPU,
                   ALL_LAYOUT,
                   phi::CholeskySolveKernel,
                   float,
                   double) {}

#endif  // not PADDLE_WITH_HIP
