#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/interpolate_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/fast_divmod.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/interpolate_function.h"

namespace phi {
using paddle::platform::FastDivMod;

template <typename T>
__forceinline__ __device__ void PreCalculatorForLinearInterpInputIndex(
    int* in_img_idx,
    int* x_id,
    T* lambda1,
    T* lambda2,
    T src_x,
    const int in_img_x) {
  src_x = (src_x > static_cast<T>(0)) ? src_x : static_cast<T>(0);
  *in_img_idx = static_cast<int>(src_x);
  *x_id = (*in_img_idx < in_img_x - 1) ? 1 : 0;
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;
  *lambda1 = static_cast<T>(static_cast<MT>(src_x) - *in_img_idx);
  *lambda2 = static_cast<T>(1.0) - *lambda1;
}

template <typename T>
__global__ void KeLinearInterpFw(const T* in,
                                 const size_t in_img_w,
                                 const size_t input_w,
                                 T* out,
                                 const size_t out_img_w,
                                 const size_t output_h,
                                 const size_t output_w,
                                 const size_t num_channels,
                                 const float ratio_w,
                                 const bool align_corners,
                                 const int align_mode,
                                 const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;  // w
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;  // w_id
    using MT = typename phi::dtype::MPTypeTrait<T>::Type;
    T src_w = static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    src_w = (src_w > static_cast<T>(0)) ? src_w : static_cast<T>(0);
    T w1lambda = align_flag
                     ? static_cast<T>(static_cast<MT>(src_w) - in_img_idx)
                     : static_cast<T>(ratio_w * out_img_idx - in_img_idx);
    T w2lambda = static_cast<T>(1.0) - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      const T* in_pos =
          &in[out_id_h * out_id_w + channel_id * in_img_size + in_img_idx];
      // linear interpolation
      out[out_id_h * output_w + out_id_w] =
          w2lambda * in_pos[0] + w1lambda * in_pos[w_id];

    } else {
      const T* in_pos =
          &in[out_id_h * input_w + in_img_idx * num_channels + channel_id];
      // linear interpolation
      out[out_id_h * output_w + out_id_w] =
          w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels];
    }
  }
}

template <typename T>
__global__ void KeNearestNeighborInterpNCHWFw(const T* in,
                                              const size_t in_img_h,
                                              const size_t in_img_w,
                                              T* out,
                                              const size_t out_img_h,
                                              const size_t out_img_w,
                                              const size_t nc,
                                              const float ratio_h,
                                              const float ratio_w,
                                              const bool align_corners) {
  int out_img_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int out_img_idy = threadIdx.y + blockIdx.y * blockDim.y;
  int nc_id = threadIdx.z + blockIdx.z * blockDim.z;
  int nc_stride = blockDim.z * gridDim.z;

  // nearest_sampling by multiple read in_addr and write to out_addr
  int in_img_idx = (align_corners)
                       ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                       : static_cast<int>(ratio_w * out_img_idx);
  int in_img_idy = (align_corners)
                       ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                       : static_cast<int>(ratio_h * out_img_idy);

  int in_index = (nc_id * in_img_h + in_img_idy) * in_img_w + in_img_idx;
  int in_index_stride = nc_stride * in_img_h * in_img_w;

  int out_index = (nc_id * out_img_h + out_img_idy) * out_img_w + out_img_idx;
  int out_index_stride = nc_stride * out_img_h * out_img_w;

  // prevent from multiple threads writing
  if (out_img_idx < out_img_w && out_img_idy < out_img_h) {
    while (nc_id < nc) {
      out[out_index] = in[in_index];
      in_index += in_index_stride;
      out_index += out_index_stride;
      nc_id += nc_stride;
    }
  }
}

template <typename T>
__global__ void KeNearestNeighborInterpFw(
    const T* in,
    const size_t in_img_h,
    const size_t in_img_w,
    const size_t input_h,
    const size_t input_w,
    T* out,
    const size_t out_img_h,
    const size_t out_img_w,
    const size_t output_h,
    const size_t output_w,
    const size_t num_channels,
    const float ratio_h,
    const float ratio_w,
    const bool align_corners,
    funcs::FastDivModForInterpolate divmods) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int in_img_size = in_img_h * in_img_w;
  int out_img_size = out_img_h * out_img_w;

  for (; tid < nthreads; tid += stride) {
    auto out_id_divmod = divmods.output_w_div.Divmod(tid);
    int out_id_h = out_id_divmod.val[0];
    int out_id_w = out_id_divmod.val[1];

    int channel_id = divmods.channels_div.Divmod(tid).val[1];
    auto outimg_id_divmod = divmods.output_wc_div.Divmod(out_id_w);
    int out_img_idy = outimg_id_divmod.val[0];
    int out_img_idx =
        divmods.channels_div.Divmod(outimg_id_divmod.val[1]).val[0];

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    out[tid] = in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
                  in_img_idx * num_channels + channel_id];
  }
}

template <typename T>
__global__ void KeBilinearInterpFw(const T* in,
                                   const size_t in_img_h,
                                   const size_t in_img_w,
                                   const size_t input_h,
                                   const size_t input_w,
                                   T* out,
                                   const size_t out_img_h,
                                   const size_t out_img_w,
                                   const size_t output_h,
                                   const size_t output_w,
                                   const size_t num_channels,
                                   const float ratio_h,
                                   const float ratio_w,
                                   const float align_type_value,
                                   funcs::FastDivModForInterpolate divmods) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < nthreads; tid += stride) {
    auto out_id_divmod = divmods.output_w_div.Divmod(tid);
    int out_id_h = out_id_divmod.val[0];
    int out_id_w = out_id_divmod.val[1];

    int channel_id = divmods.channels_div.Divmod(tid).val[1];
    auto outimg_id_divmod = divmods.output_wc_div.Divmod(out_id_w);
    int out_img_idy = outimg_id_divmod.val[0];
    int out_img_idx =
        divmods.channels_div.Divmod(outimg_id_divmod.val[1]).val[0];

    int in_img_idx, in_img_idy, h_id, w_id;
    T h1lambda, w1lambda, h2lambda, w2lambda;
    T src_w = static_cast<T>(ratio_w * (out_img_idx + align_type_value) -
                             align_type_value);
    T src_h = static_cast<T>(ratio_h * (out_img_idy + align_type_value) -
                             align_type_value);

    PreCalculatorForLinearInterpInputIndex(
        &in_img_idx, &w_id, &w1lambda, &w2lambda, src_w, in_img_w);
    PreCalculatorForLinearInterpInputIndex(
        &in_img_idy, &h_id, &h1lambda, &h2lambda, src_h, in_img_h);

    // bilinear interpolation
    const T* in_pos =
        &in[out_id_h * input_w + in_img_idy * in_img_w * num_channels +
            in_img_idx * num_channels + channel_id];
    out[tid] =
        h2lambda *
            (w2lambda * in_pos[0] + w1lambda * in_pos[w_id * num_channels]) +
        h1lambda *
            (w2lambda * in_pos[h_id * in_img_w * num_channels] +
             w1lambda *
                 in_pos[h_id * in_img_w * num_channels + w_id * num_channels]);
  }
}

template <typename T>
__global__ void KeBilinearInterpNCHWFw(const T* in,
                                       const size_t in_img_h,
                                       const size_t in_img_w,
                                       T* out,
                                       const size_t out_img_h,
                                       const size_t out_img_w,
                                       const size_t nc,
                                       const float ratio_h,
                                       const float ratio_w,
                                       const float align_type_value) {
  int out_img_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int out_img_idy = threadIdx.y + blockIdx.y * blockDim.y;
  int nc_id = threadIdx.z + blockIdx.z * blockDim.z;
  int nc_stride = blockDim.z * gridDim.z;

  int in_img_idx, in_img_idy, h_id, w_id;
  T h1lambda, w1lambda, h2lambda, w2lambda;
  T src_w = static_cast<T>(ratio_w * (out_img_idx + align_type_value) -
                           align_type_value);
  T src_h = static_cast<T>(ratio_h * (out_img_idy + align_type_value) -
                           align_type_value);

  PreCalculatorForLinearInterpInputIndex(
      &in_img_idx, &w_id, &w1lambda, &w2lambda, src_w, in_img_w);
  PreCalculatorForLinearInterpInputIndex(
      &in_img_idy, &h_id, &h1lambda, &h2lambda, src_h, in_img_h);

  int in_index = (nc_id * in_img_h + in_img_idy) * in_img_w + in_img_idx;
  int in_index_stride = nc_stride * in_img_h * in_img_w;

  int out_index = (nc_id * out_img_h + out_img_idy) * out_img_w + out_img_idx;
  int out_index_stride = nc_stride * out_img_h * out_img_w;

  // prevent from multiple threads writing
  if (out_img_idx < out_img_w && out_img_idy < out_img_h) {
    while (nc_id < nc) {
      const T* in_pos = &in[in_index];
      out[out_index] =
          h2lambda * (w2lambda * in_pos[0] + w1lambda * in_pos[w_id]) +
          h1lambda * (w2lambda * in_pos[h_id * in_img_w] +
                      w1lambda * in_pos[h_id * in_img_w + w_id]);

      in_index += in_index_stride;
      out_index += out_index_stride;
      nc_id += nc_stride;
    }
  }
}

template <typename T>
__device__ __forceinline__ static T Kecubic_interp(
    const T x0, const T x1, const T x2, const T x3, T t) {
  T coeffs[4];
  T a = static_cast<T>(-0.75);
  T x_1 = t;
  T x_2 = static_cast<T>(1.0) - t;
  coeffs[0] = funcs::CubicConvolution2<T>(x_1 + static_cast<T>(1.0), a);
  coeffs[1] = funcs::CubicConvolution1<T>(x_1, a);
  coeffs[2] = funcs::CubicConvolution1<T>(x_2, a);
  coeffs[3] = funcs::CubicConvolution2<T>(x_2 + static_cast<T>(1.0), a);
  return x0 * coeffs[0] + x1 * coeffs[1] + x2 * coeffs[2] + x3 * coeffs[3];
}

template <typename T>
__global__ void KeBicubicInterpFw(const T* in,
                                  const size_t in_img_h,
                                  const size_t in_img_w,
                                  const size_t input_h,
                                  const size_t input_w,
                                  T* out,
                                  const size_t out_img_h,
                                  const size_t out_img_w,
                                  const size_t output_h,
                                  const size_t output_w,
                                  const size_t num_channels,
                                  const float ratio_h,
                                  const float ratio_w,
                                  const bool align_corners,
                                  const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idy, out_img_idx;

    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idy = (out_id_w % out_img_size) / out_img_w;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idy = out_id_w / (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    T in_img_idy = align_corners
                       ? static_cast<T>(ratio_h * out_img_idy)
                       : static_cast<T>(ratio_h * (out_img_idy + 0.5) - 0.5);
    int input_y = floorf(in_img_idy);
    using MT = typename phi::dtype::MPTypeTrait<T>::Type;
    const T y_t = static_cast<T>(static_cast<MT>(in_img_idy) - input_y);

    T in_img_idx = align_corners
                       ? static_cast<T>(ratio_w * out_img_idx)
                       : static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    int input_x = floorf(in_img_idx);
    const T x_t = static_cast<T>(static_cast<MT>(in_img_idx) - input_x);

    T coefficients[4];
    const T* in_pos_0;
    const T* in_pos_1;
    const T* in_pos_2;
    const T* in_pos_3;
    int access_x_0;
    if (data_layout == DataLayout::kNCHW) {
      for (int k = 0; k < 4; k++) {
        int access_y =
            max(min(input_y - 1 + k, static_cast<int>(in_img_h - 1)), 0);
        access_x_0 = max(min(input_x - 1, static_cast<int>(in_img_w - 1)), 0);
        int access_x_1 =
            max(min(input_x + 0, static_cast<int>(in_img_w - 1)), 0);
        int access_x_2 =
            max(min(input_x + 1, static_cast<int>(in_img_w - 1)), 0);
        int access_x_3 =
            max(min(input_x + 2, static_cast<int>(in_img_w - 1)), 0);

        in_pos_0 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_0];
        in_pos_1 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_1];
        in_pos_2 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_2];
        in_pos_3 = &in[out_id_h * input_w + channel_id * in_img_size +
                       access_y * in_img_w + access_x_3];

        coefficients[k] = Kecubic_interp<T>(
            in_pos_0[0], in_pos_1[0], in_pos_2[0], in_pos_3[0], x_t);
      }

      out[out_id_h * output_w + out_id_w] = Kecubic_interp<T>(coefficients[0],
                                                              coefficients[1],
                                                              coefficients[2],
                                                              coefficients[3],
                                                              y_t);

    } else {
      for (int k = 0; k < 4; k++) {
        int access_y =
            max(min(input_y - 1 + k, static_cast<int>((in_img_h - 1))), 0);
        int access_x_0 =
            max(min(input_x - 1, static_cast<int>((in_img_w - 1))), 0);
        int access_x_1 =
            max(min(input_x + 0, static_cast<int>((in_img_w - 1))), 0);
        int access_x_2 =
            max(min(input_x + 1, static_cast<int>((in_img_w - 1))), 0);
        int access_x_3 =
            max(min(input_x + 2, static_cast<int>((in_img_w - 1))), 0);

        const T* in_pos_0 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_0 * num_channels + channel_id];
        const T* in_pos_1 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_1 * num_channels + channel_id];
        const T* in_pos_2 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_2 * num_channels + channel_id];
        const T* in_pos_3 =
            &in[out_id_h * input_w + access_y * in_img_w * num_channels +
                access_x_3 * num_channels + channel_id];

        coefficients[k] = Kecubic_interp<T>(
            in_pos_0[0], in_pos_1[0], in_pos_2[0], in_pos_3[0], x_t);
      }

      out[out_id_h * output_w + out_id_w] = Kecubic_interp<T>(coefficients[0],
                                                              coefficients[1],
                                                              coefficients[2],
                                                              coefficients[3],
                                                              y_t);
    }
  }
}

template <typename T>
__global__ void KeTrilinearInterpFw(const T* in,
                                    const size_t in_img_d,
                                    const size_t in_img_h,
                                    const size_t in_img_w,
                                    const size_t input_h,
                                    const size_t input_w,
                                    T* out,
                                    const size_t out_img_d,
                                    const size_t out_img_h,
                                    const size_t out_img_w,
                                    const size_t output_h,
                                    const size_t output_w,
                                    const size_t num_channels,
                                    const float ratio_d,
                                    const float ratio_h,
                                    const float ratio_w,
                                    const bool align_corners,
                                    const int align_mode,
                                    const DataLayout data_layout) {
  int nthreads = output_h * output_w;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  bool align_flag = (align_mode == 0 && !align_corners);
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = align_flag
                         ? static_cast<int>(ratio_d * (out_img_idt + 0.5) - 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);
    in_img_idt = (in_img_idt > 0) ? in_img_idt : 0;
    int d_id = (in_img_idt < in_img_d - 1) ? 1 : 0;
    using MT = typename phi::dtype::MPTypeTrait<T>::Type;
    T src_d = static_cast<T>(ratio_d * (out_img_idt + 0.5) - 0.5);
    src_d = (src_d > static_cast<T>(0)) ? src_d : static_cast<T>(0);
    T d1lambda = align_flag
                     ? static_cast<T>(static_cast<MT>(src_d) - in_img_idt)
                     : static_cast<T>(ratio_d * out_img_idt - in_img_idt);
    T d2lambda = static_cast<T>(1.0) - d1lambda;

    int in_img_idy = align_flag
                         ? static_cast<int>(ratio_h * (out_img_idy + 0.5) - 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    in_img_idy = (in_img_idy > 0) ? in_img_idy : 0;
    int h_id = (in_img_idy < in_img_h - 1) ? 1 : 0;
    T src_h = static_cast<T>(ratio_h * (out_img_idy + 0.5) - 0.5);
    src_h = (src_h > static_cast<T>(0)) ? src_h : static_cast<T>(0);
    T h1lambda = align_flag
                     ? static_cast<T>(static_cast<MT>(src_h) - in_img_idy)
                     : static_cast<T>(ratio_h * out_img_idy - in_img_idy);
    T h2lambda = static_cast<T>(1.0) - h1lambda;

    int in_img_idx = align_flag
                         ? static_cast<int>(ratio_w * (out_img_idx + 0.5) - 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);
    in_img_idx = (in_img_idx > 0) ? in_img_idx : 0;
    int w_id = (in_img_idx < in_img_w - 1) ? 1 : 0;
    T src_w = static_cast<T>(ratio_w * (out_img_idx + 0.5) - 0.5);
    src_w = (src_w > static_cast<T>(0)) ? src_w : static_cast<T>(0);
    T w1lambda = align_flag
                     ? static_cast<T>(static_cast<MT>(src_w) - in_img_idx)
                     : static_cast<T>(ratio_w * out_img_idx - in_img_idx);
    T w2lambda = static_cast<T>(1.0) - w1lambda;

    if (data_layout == DataLayout::kNCHW) {
      int in_pos1_idx = out_id_h * input_w + channel_id * in_img_size +
                        (in_img_idt * in_img_h + in_img_idy) * in_img_w +
                        in_img_idx;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] + w1lambda * in_pos1[w_id]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w] +
                           w1lambda * in_pos1[h_id * in_img_w + w_id])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] + w1lambda * in_pos2[w_id]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w] +
                           w1lambda * in_pos2[h_id * in_img_w + w_id]));

    } else {
      int in_pos1_idx = out_id_h * input_w +
                        in_img_idt * in_img_h * in_img_w * num_channels +
                        in_img_idy * in_img_w * num_channels +
                        in_img_idx * num_channels + channel_id;
      const T* in_pos1 = &in[in_pos1_idx];
      int in_pos2_idx = in_pos1_idx + d_id * in_img_h * in_img_w * num_channels;
      const T* in_pos2 = &in[in_pos2_idx];

      // trilinear interpolation
      out[out_id_h * output_w + out_id_w] =
          d2lambda *
              (h2lambda * (w2lambda * in_pos1[0] +
                           w1lambda * in_pos1[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos1[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos1[h_id * in_img_w * num_channels +
                                              w_id * num_channels])) +
          d1lambda *
              (h2lambda * (w2lambda * in_pos2[0] +
                           w1lambda * in_pos2[w_id * num_channels]) +
               h1lambda * (w2lambda * in_pos2[h_id * in_img_w * num_channels] +
                           w1lambda * in_pos2[h_id * in_img_w * num_channels +
                                              w_id * num_channels]));
    }
  }
}

template <typename T>
__global__ void KeNearestNeighbor3DInterpFw(const T* in,
                                            const size_t in_img_d,
                                            const size_t in_img_h,
                                            const size_t in_img_w,
                                            const size_t input_h,
                                            const size_t input_w,
                                            T* out,
                                            const size_t out_img_d,
                                            const size_t out_img_h,
                                            const size_t out_img_w,
                                            const size_t output_h,
                                            const size_t output_w,
                                            const size_t num_channels,
                                            const float ratio_d,
                                            const float ratio_h,
                                            const float ratio_w,
                                            const bool align_corners,
                                            const DataLayout data_layout) {
  int nthreads = output_h * output_w;  // ncdhw
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (; tid < nthreads; tid += stride) {
    int out_id_h = tid / output_w;
    int out_id_w = tid % output_w;
    int in_img_size = input_w / num_channels;
    int out_img_size = output_w / num_channels;

    int channel_id, out_img_idt, out_img_idy, out_img_idx;
    if (data_layout == DataLayout::kNCHW) {
      channel_id = out_id_w / out_img_size;
      out_img_idt = (out_id_w % out_img_size) / out_img_h / out_img_w;
      out_img_idy = ((out_id_w % out_img_size) / out_img_w) % out_img_h;
      out_img_idx = tid % out_img_w;
    } else {
      out_img_idt = out_id_w / (out_img_h * out_img_w * num_channels);
      out_img_idy = out_id_w % (out_img_h * out_img_w * num_channels) /
                    (out_img_w * num_channels);
      out_img_idx = out_id_w % (out_img_w * num_channels) / num_channels;
      channel_id = tid % num_channels;
    }

    int in_img_idt = (align_corners)
                         ? static_cast<int>(ratio_d * out_img_idt + 0.5)
                         : static_cast<int>(ratio_d * out_img_idt);

    int in_img_idy = (align_corners)
                         ? static_cast<int>(ratio_h * out_img_idy + 0.5)
                         : static_cast<int>(ratio_h * out_img_idy);
    int in_img_idx = (align_corners)
                         ? static_cast<int>(ratio_w * out_img_idx + 0.5)
                         : static_cast<int>(ratio_w * out_img_idx);

    if (data_layout == DataLayout::kNCHW) {
      out[tid] = in[out_id_h * input_w + channel_id * in_img_size +
                    in_img_idt * in_img_h * in_img_w + in_img_idy * in_img_w +
                    in_img_idx];
    } else {
      out[tid] = in[out_id_h * input_w +
                    in_img_idt * in_img_h * in_img_w * num_channels +
                    in_img_idy * in_img_w * num_channels +
                    in_img_idx * num_channels + channel_id];
    }
  }
}

template <typename T, typename Context>
static void Interpolate1DCUDAFwd(
    const Context& dev_ctx,
    const DenseTensor& input,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout_str,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  auto* input_data = input.data<T>();

  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  funcs::ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  float scale_w = -1;
  if (size_tensor && size_tensor->size() > 0) {
    // have size tensor
    auto new_size = funcs::get_new_shape(size_tensor.get());
    out_w = new_size[0];
  } else {
    if (scale_tensor) {
      auto scale_data =
          funcs::get_new_data_from_tensor<float>(scale_tensor.get_ptr());
      scale_w = scale_data[0];
      PADDLE_ENFORCE_EQ(
          scale_w > 0,
          true,
          errors::InvalidArgument(
              "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
    } else {
      if (scale.size() > 0) {
        scale_w = scale[0];
        PADDLE_ENFORCE_EQ(
            scale_w > 0,
            true,
            errors::InvalidArgument(
                "The scale_w in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_w));
      }
    }
    if (scale_w > 0.) {
      out_w = static_cast<int>(in_w * scale_w);
    }
    if (out_size) {
      DenseTensor sizes;
      paddle::framework::TensorCopySync(
          *out_size, paddle::platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_w = size_data[0];
    }
  }
  PADDLE_ENFORCE_GT(
      out_w,
      0,
      errors::InvalidArgument("out_w in Attr(out_shape) of Op(interpolate) "
                              "should be greater than 0."));
  phi::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_w};
  } else {
    dim_out = {n, out_w, c};
  }
  output->Resize(dim_out);
  auto output_data = dev_ctx.template Alloc<T>(output);

  if (in_w == out_w) {
    paddle::framework::TensorCopy(input, dev_ctx.GetPlace(), output);
    return;
  }

  float ratio_w = 0.f;
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1.0) / (out_w - 1.0)
                              : static_cast<float>(new_scale_w);
  }

  int64_t in_cw = c * in_w;
  int64_t out_cw = c * out_w;
  auto pixelNum = n * out_cw;

  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, pixelNum);

  if ("linear" == interp_method) {
    KeLinearInterpFw<T><<<config.block_per_grid,
                          config.thread_per_block,
                          0,
                          dev_ctx.stream()>>>(input_data,
                                              in_w,
                                              in_cw,
                                              output_data,
                                              out_w,
                                              n,
                                              out_cw,
                                              c,
                                              ratio_w,
                                              align_corners,
                                              align_mode,
                                              data_layout);
  }
}

template <typename T, typename Context>
static void Interpolate2DCUDAFwd(
    const Context& dev_ctx,
    const DenseTensor& input,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout_str,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  auto* input_data = input.data<T>();

  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  funcs::ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  float scale_w = -1;
  float scale_h = -1;
  if (size_tensor && size_tensor->size() > 0) {
    // have size tensor
    auto new_size = funcs::get_new_shape(size_tensor.get());
    out_h = new_size[0];
    out_w = new_size[1];
  } else {
    if (scale_tensor) {
      auto scale_data =
          funcs::get_new_data_from_tensor<float>(scale_tensor.get_ptr());
      if (scale_data.size() > 1) {
        scale_h = scale_data[0];
        scale_w = scale_data[1];
      } else {
        scale_h = scale_data[0];
        scale_w = scale_data[0];
      }

      PADDLE_ENFORCE_EQ(
          scale_w > 0,
          true,
          errors::InvalidArgument(
              "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
      PADDLE_ENFORCE_EQ(
          scale_h > 0,
          true,
          errors::InvalidArgument(
              "The scale_h in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_h));
    } else {
      if (scale.size() > 1) {
        scale_w = scale[1];
        scale_h = scale[0];

        PADDLE_ENFORCE_EQ(
            scale_w > 0,
            true,
            errors::InvalidArgument(
                "The scale_w in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_w));
        PADDLE_ENFORCE_EQ(
            scale_h > 0,
            true,
            errors::InvalidArgument(
                "The scale_h in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_h));
      }
    }
    if (scale_w > 0. && scale_h > 0.) {
      out_h = static_cast<int>(in_h * scale_h);
      out_w = static_cast<int>(in_w * scale_w);
    }
    if (out_size) {
      DenseTensor sizes;
      paddle::framework::TensorCopySync(
          *out_size, paddle::platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_h = size_data[0];
      out_w = size_data[1];
    }
  }
  PADDLE_ENFORCE_GT(
      out_h,
      0,
      errors::InvalidArgument("out_h in Attr(out_shape) of Op(interpolate) "
                              "should be greater than 0."));
  PADDLE_ENFORCE_GT(
      out_w,
      0,
      errors::InvalidArgument("out_w in Attr(out_shape) of Op(interpolate) "
                              "should be greater than 0."));

  phi::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_h, out_w};
  } else {
    dim_out = {n, out_h, out_w, c};
  }
  output->Resize(dim_out);
  auto output_data = dev_ctx.template Alloc<T>(output);

  if (in_h == out_h && in_w == out_w) {
    paddle::framework::TensorCopy(input, dev_ctx.GetPlace(), output);
    return;
  }

  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_h > 1) {
    float new_scale_h = 0.f;
    new_scale_h = (scale_h > 0) ? static_cast<float>(1. / scale_h)
                                : static_cast<float>(in_h) / out_h;
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(new_scale_h);
  }
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }

  int64_t in_hw = in_h * in_w;
  int64_t out_hw = out_h * out_w;
  int64_t in_chw = c * in_hw;
  int64_t out_chw = c * out_hw;

  auto pixelNum = n * out_chw;

  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, pixelNum);

  if ("nearest" == interp_method) {
    if (data_layout == DataLayout::kNCHW) {
      // get launch 3D config
      int nc = n * c;
      backends::gpu::GpuLaunchConfig config_3d =
          backends::gpu::GetGpuLaunchConfig3D(dev_ctx, nc, out_h, out_w);
      KeNearestNeighborInterpNCHWFw<T><<<config_3d.block_per_grid,
                                         config_3d.thread_per_block,
                                         0,
                                         dev_ctx.stream()>>>(input_data,
                                                             in_h,
                                                             in_w,
                                                             output_data,
                                                             out_h,
                                                             out_w,
                                                             nc,
                                                             ratio_h,
                                                             ratio_w,
                                                             align_corners);
    } else {
      int64_t cw = c * out_w;
      auto interp_divmods = funcs::FastDivModForInterpolate(c, out_chw, cw);
      KeNearestNeighborInterpFw<T><<<config.block_per_grid,
                                     config.thread_per_block,
                                     0,
                                     dev_ctx.stream()>>>(input_data,
                                                         in_h,
                                                         in_w,
                                                         n,
                                                         in_chw,
                                                         output_data,
                                                         out_h,
                                                         out_w,
                                                         n,
                                                         out_chw,
                                                         c,
                                                         ratio_h,
                                                         ratio_w,
                                                         align_corners,
                                                         interp_divmods);
    }
  } else if ("bilinear" == interp_method) {
    dim3 thread_num = config.thread_per_block;
#ifdef WITH_NV_JETSON
    if (config.compute_capability == 53 || config.compute_capability == 62) {
      thread_num = 512;
    }
#endif
    const float align_type_value =
        (align_mode == 0 && !align_corners) ? 0.5f : 0.f;
    if (data_layout == DataLayout::kNCHW) {
      // get launch 3D config
      int nc = n * c;
      backends::gpu::GpuLaunchConfig config_3d =
          backends::gpu::GetGpuLaunchConfig3D(dev_ctx, nc, out_h, out_w);
      KeBilinearInterpNCHWFw<T><<<config_3d.block_per_grid,
                                  config_3d.thread_per_block,
                                  0,
                                  dev_ctx.stream()>>>(input_data,
                                                      in_h,
                                                      in_w,
                                                      output_data,
                                                      out_h,
                                                      out_w,
                                                      nc,
                                                      ratio_h,
                                                      ratio_w,
                                                      align_type_value);
    } else {
      int64_t cw = c * out_w;
      auto interp_divmods = funcs::FastDivModForInterpolate(c, out_chw, cw);
      KeBilinearInterpFw<T>
          <<<config.block_per_grid, thread_num, 0, dev_ctx.stream()>>>(
              input_data,
              in_h,
              in_w,
              n,
              in_chw,
              output_data,
              out_h,
              out_w,
              n,
              out_chw,
              c,
              ratio_h,
              ratio_w,
              align_type_value,
              interp_divmods);
    }
  } else if ("bicubic" == interp_method) {
#ifdef __HIPCC__
    constexpr int thread_per_block = 256;
#else
    constexpr int thread_per_block = 512;
#endif
    KeBicubicInterpFw<T>
        <<<config.block_per_grid, thread_per_block, 0, dev_ctx.stream()>>>(
            input_data,
            in_h,
            in_w,
            n,
            in_chw,
            output_data,
            out_h,
            out_w,
            n,
            out_chw,
            c,
            ratio_h,
            ratio_w,
            align_corners,
            data_layout);
  }
}

template <typename T, typename Context>
static void Interpolate3DCUDAFwd(
    const Context& dev_ctx,
    const DenseTensor& input,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout_str,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  auto* input_data = input.data<T>();

  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_layout_str);
  int n, c, in_d, in_h, in_w;
  funcs::ExtractNCDWH(input.dims(), data_layout, &n, &c, &in_d, &in_h, &in_w);

  float scale_w = -1;
  float scale_d = -1;
  float scale_h = -1;
  if (size_tensor && size_tensor->size() > 0) {
    // have size tensor
    auto new_size = funcs::get_new_shape(size_tensor.get());
    out_d = new_size[0];
    out_h = new_size[1];
    out_w = new_size[2];
  } else {
    if (scale_tensor) {
      auto scale_data =
          funcs::get_new_data_from_tensor<float>(scale_tensor.get_ptr());
      if (scale_data.size() > 2) {
        scale_d = scale_data[0];
        scale_h = scale_data[1];
        scale_w = scale_data[2];
      } else {
        scale_d = scale_data[0];
        scale_h = scale_data[0];
        scale_w = scale_data[0];
      }

      PADDLE_ENFORCE_EQ(
          scale_w > 0,
          true,
          errors::InvalidArgument(
              "The scale_w in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_w));
      PADDLE_ENFORCE_EQ(
          scale_h > 0,
          true,
          errors::InvalidArgument(
              "The scale_h in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_h));
      PADDLE_ENFORCE_EQ(
          scale_d > 0,
          true,
          errors::InvalidArgument(
              "The scale_d in input 'Scale' Tensor of Operator(interpolate) "
              "should be greater than 0, but received value is %d.",
              scale_d));
    } else {
      if (scale.size() > 2) {
        scale_d = scale[0];
        scale_h = scale[1];
        scale_w = scale[2];

        PADDLE_ENFORCE_EQ(
            scale_w > 0,
            true,
            errors::InvalidArgument(
                "The scale_w in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_w));
        PADDLE_ENFORCE_EQ(
            scale_h > 0,
            true,
            errors::InvalidArgument(
                "The scale_h in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_h));
        PADDLE_ENFORCE_EQ(
            scale_d > 0,
            true,
            errors::InvalidArgument(
                "The scale_d in Attr(scale) of Operator(interpolate) "
                "should be greater than 0, but received value is %d.",
                scale_d));
      }
    }
    if (scale_d > 0. && scale_h > 0. && scale_w > 0.) {
      out_d = static_cast<int>(in_d * scale_d);
      out_h = static_cast<int>(in_h * scale_h);
      out_w = static_cast<int>(in_w * scale_w);
    }
    if (out_size) {
      DenseTensor sizes;
      paddle::framework::TensorCopySync(
          *out_size, paddle::platform::CPUPlace(), &sizes);
      auto size_data = sizes.data<int>();
      out_d = size_data[0];
      out_h = size_data[1];
      out_w = size_data[2];
    }
  }
  PADDLE_ENFORCE_GT(
      out_d,
      0,
      errors::InvalidArgument("out_d in Attr(out_shape) of Op(interpolate) "
                              "should be greater than 0."));
  PADDLE_ENFORCE_GT(
      out_h,
      0,
      errors::InvalidArgument("out_h in Attr(out_shape) of Op(interpolate) "
                              "should be greater than 0."));
  PADDLE_ENFORCE_GT(
      out_w,
      0,
      errors::InvalidArgument("out_w in Attr(out_shape) of Op(interpolate) "
                              "should be greater than 0."));

  phi::DDim dim_out;
  if (data_layout == DataLayout::kNCHW) {
    dim_out = {n, c, out_d, out_h, out_w};
  } else {
    dim_out = {n, out_d, out_h, out_w, c};
  }
  output->Resize(dim_out);
  auto output_data = dev_ctx.template Alloc<T>(output);

  if (in_d == out_d && in_h == out_h && in_w == out_w) {
    paddle::framework::TensorCopy(input, dev_ctx.GetPlace(), output);
    return;
  }

  float ratio_d = 0.f;
  float ratio_h = 0.f;
  float ratio_w = 0.f;
  if (out_d > 1) {
    float new_scale_d = 0.f;
    new_scale_d = (scale_d > 0) ? static_cast<float>(1. / scale_d)
                                : static_cast<float>(in_d) / out_d;
    ratio_d = (align_corners) ? static_cast<float>(in_d - 1) / (out_d - 1)
                              : static_cast<float>(new_scale_d);
  }
  if (out_h > 1) {
    float new_scale_h = 0.f;
    new_scale_h = (scale_h > 0) ? static_cast<float>(1. / scale_h)
                                : static_cast<float>(in_h) / out_h;
    ratio_h = (align_corners) ? static_cast<float>(in_h - 1) / (out_h - 1)
                              : static_cast<float>(new_scale_h);
  }
  if (out_w > 1) {
    float new_scale_w = 0.f;
    new_scale_w = (scale_w > 0) ? static_cast<float>(1. / scale_w)
                                : static_cast<float>(in_w) / out_w;
    ratio_w = (align_corners) ? static_cast<float>(in_w - 1) / (out_w - 1)
                              : static_cast<float>(new_scale_w);
  }

  int64_t in_dhw = in_d * in_h * in_w;
  int64_t out_dhw = out_d * out_h * out_w;
  int64_t in_cdhw = c * in_dhw;
  int64_t out_cdhw = c * out_dhw;

  auto pixelNum = n * out_cdhw;

  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, pixelNum);

  if ("trilinear" == interp_method) {
    KeTrilinearInterpFw<T><<<config.block_per_grid,
                             config.thread_per_block,
                             0,
                             dev_ctx.stream()>>>(input_data,
                                                 in_d,
                                                 in_h,
                                                 in_w,
                                                 n,
                                                 in_cdhw,
                                                 output_data,
                                                 out_d,
                                                 out_h,
                                                 out_w,
                                                 n,
                                                 out_cdhw,
                                                 c,
                                                 ratio_d,
                                                 ratio_h,
                                                 ratio_w,
                                                 align_corners,
                                                 align_mode,
                                                 data_layout);
  } else if ("nearest" == interp_method) {
    KeNearestNeighbor3DInterpFw<T><<<config.block_per_grid,
                                     config.thread_per_block,
                                     0,
                                     dev_ctx.stream()>>>(input_data,
                                                         in_d,
                                                         in_h,
                                                         in_w,
                                                         n,
                                                         in_cdhw,
                                                         output_data,
                                                         out_d,
                                                         out_h,
                                                         out_w,
                                                         n,
                                                         out_cdhw,
                                                         c,
                                                         ratio_d,
                                                         ratio_h,
                                                         ratio_w,
                                                         align_corners,
                                                         data_layout);
  }
}

template <typename T, typename Context>
void InterpolateKernel(
    const Context& dev_ctx,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  auto input_dims = x.dims();
  if (input_dims.size() == 3) {  // 1D interpolation
    Interpolate1DCUDAFwd<T, Context>(dev_ctx,
                                     x,
                                     out_size,
                                     size_tensor,
                                     scale_tensor,
                                     data_layout,
                                     out_w,
                                     scale,
                                     interp_method,
                                     align_corners,
                                     align_mode,
                                     output);
  } else if (input_dims.size() == 4) {  // 2D interpolation
    Interpolate2DCUDAFwd<T, Context>(dev_ctx,
                                     x,
                                     out_size,
                                     size_tensor,
                                     scale_tensor,
                                     data_layout,
                                     out_h,
                                     out_w,
                                     scale,
                                     interp_method,
                                     align_corners,
                                     align_mode,
                                     output);
  } else if (input_dims.size() == 5) {  // 3D interpolation
    Interpolate3DCUDAFwd<T, Context>(dev_ctx,
                                     x,
                                     out_size,
                                     size_tensor,
                                     scale_tensor,
                                     data_layout,
                                     out_d,
                                     out_h,
                                     out_w,
                                     scale,
                                     interp_method,
                                     align_corners,
                                     align_mode,
                                     output);
  }
}

template <typename T, typename Context>
void BilinearInterpKernel(
    const Context& dev_ctx,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  InterpolateKernel<T, Context>(dev_ctx,
                                x,
                                out_size,
                                size_tensor,
                                scale_tensor,
                                data_layout,
                                out_d,
                                out_h,
                                out_w,
                                scale,
                                interp_method,
                                align_corners,
                                align_mode,
                                output);
}

template <typename T, typename Context>
void NearestInterpKernel(
    const Context& dev_ctx,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  InterpolateKernel<T, Context>(dev_ctx,
                                x,
                                out_size,
                                size_tensor,
                                scale_tensor,
                                data_layout,
                                out_d,
                                out_h,
                                out_w,
                                scale,
                                interp_method,
                                align_corners,
                                align_mode,
                                output);
}

template <typename T, typename Context>
void TrilinearInterpKernel(
    const Context& dev_ctx,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  InterpolateKernel<T, Context>(dev_ctx,
                                x,
                                out_size,
                                size_tensor,
                                scale_tensor,
                                data_layout,
                                out_d,
                                out_h,
                                out_w,
                                scale,
                                interp_method,
                                align_corners,
                                align_mode,
                                output);
}

template <typename T, typename Context>
void LinearInterpKernel(
    const Context& dev_ctx,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  InterpolateKernel<T, Context>(dev_ctx,
                                x,
                                out_size,
                                size_tensor,
                                scale_tensor,
                                data_layout,
                                out_d,
                                out_h,
                                out_w,
                                scale,
                                interp_method,
                                align_corners,
                                align_mode,
                                output);
}

template <typename T, typename Context>
void BicubicInterpKernel(
    const Context& dev_ctx,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& out_size,
    const paddle::optional<std::vector<const DenseTensor*>>& size_tensor,
    const paddle::optional<DenseTensor>& scale_tensor,
    const std::string& data_layout,
    int out_d,
    int out_h,
    int out_w,
    const std::vector<float>& scale,
    const std::string& interp_method,
    bool align_corners,
    int align_mode,
    DenseTensor* output) {
  InterpolateKernel<T, Context>(dev_ctx,
                                x,
                                out_size,
                                size_tensor,
                                scale_tensor,
                                data_layout,
                                out_d,
                                out_h,
                                out_w,
                                scale,
                                interp_method,
                                align_corners,
                                align_mode,
                                output);
}

}  // namespace phi

PD_REGISTER_KERNEL(bilinear_interp,
                   GPU,
                   ALL_LAYOUT,
                   phi::BilinearInterpKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int) {
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(3).SetBackend(phi::Backend::ALL_BACKEND);
}
PD_REGISTER_KERNEL(nearest_interp,
                   GPU,
                   ALL_LAYOUT,
                   phi::NearestInterpKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int,
                   int64_t) {
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(3).SetBackend(phi::Backend::ALL_BACKEND);
}
PD_REGISTER_KERNEL(trilinear_interp,
                   GPU,
                   ALL_LAYOUT,
                   phi::TrilinearInterpKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int) {
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(3).SetBackend(phi::Backend::ALL_BACKEND);
}
PD_REGISTER_KERNEL(linear_interp,
                   GPU,
                   ALL_LAYOUT,
                   phi::LinearInterpKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int) {
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(3).SetBackend(phi::Backend::ALL_BACKEND);
}
PD_REGISTER_KERNEL(bicubic_interp,
                   GPU,
                   ALL_LAYOUT,
                   phi::BicubicInterpKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int) {
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(3).SetBackend(phi::Backend::ALL_BACKEND);
}
