#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/temporal_shift_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T>
__global__ void KeTemporalShiftFwNCHW(const T* input,
                                      T* output,
                                      const int ntchw,
                                      const int tchw,
                                      const int chw,
                                      const int hw,
                                      const int t,
                                      const int c1,
                                      const int c2) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int src_it = 0;

  for (; tid < ntchw; tid += stride) {
    int it = (tid % tchw) / chw;
    int ic = (tid % chw) / hw;

    if (ic < c1) {
      src_it = it - 1;
    } else if (ic < c2) {
      src_it = it + 1;
    } else {
      src_it = it;
    }

    if (src_it < 0 || src_it >= t) {
      output[tid] = 0;
    } else {
      output[tid] = input[tid + (src_it - it) * chw];
    }
  }
}

template <typename T>
__global__ void KeTemporalShiftFwNHWC(const T* input,
                                      T* output,
                                      const int nthwc,
                                      const int thwc,
                                      const int hwc,
                                      const int t,
                                      const int c,
                                      const int c1,
                                      const int c2) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int src_it = 0;

  for (; tid < nthwc; tid += stride) {
    int it = (tid % thwc) / hwc;
    int ic = tid % c;

    if (ic < c1) {
      src_it = it - 1;
    } else if (ic < c2) {
      src_it = it + 1;
    } else {
      src_it = it;
    }

    if (src_it < 0 || src_it >= t) {
      output[tid] = 0;
    } else {
      output[tid] = input[tid + (src_it - it) * hwc];
    }
  }
}

template <typename T, typename Context>
void TemporalShiftKernel(const Context& dev_ctx,
                         const DenseTensor& x,
                         int seg_num,
                         float shift_ratio,
                         const std::string& data_format_str,
                         DenseTensor* out) {
  auto* input = &x;
  auto* output = out;
  int t = seg_num;
  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_format_str);

  const int nt = input->dims()[0];
  const int c =
      (data_layout == DataLayout::kNCHW ? input->dims()[1] : input->dims()[3]);
  const int h =
      (data_layout == DataLayout::kNCHW ? input->dims()[2] : input->dims()[1]);
  const int w =
      (data_layout == DataLayout::kNCHW ? input->dims()[3] : input->dims()[2]);

  const int hw = h * w;
  const int chw = c * hw;
  const int tchw = t * chw;
  const int ntchw = nt * chw;

  const int c1 = static_cast<int>(c * shift_ratio);
  const int c2 = static_cast<int>(c * 2 * shift_ratio);

  DDim out_dims =
      (data_layout == DataLayout::kNCHW ? phi::make_ddim({nt, c, h, w})
                                        : phi::make_ddim({nt, h, w, c}));
  const T* input_data = input->data<T>();
  T* output_data = output->mutable_data<T>(out_dims, dev_ctx.GetPlace());

  int pixelNum = nt * chw;
  int threads = 1024;
  int grid = (pixelNum + threads - 1) / threads;
  int blocks_per_sm = dev_ctx.GetMaxPhysicalThreadCount() / threads;
  grid = std::min(dev_ctx.GetSMCount() * blocks_per_sm, grid);

  if (data_layout == DataLayout::kNCHW) {
    KeTemporalShiftFwNCHW<T><<<grid, threads, 0, dev_ctx.stream()>>>(
        input_data, output_data, ntchw, tchw, chw, hw, t, c1, c2);
  } else {
    KeTemporalShiftFwNHWC<T><<<grid, threads, 0, dev_ctx.stream()>>>(
        input_data, output_data, ntchw, tchw, chw, t, c, c1, c2);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(temporal_shift,
                   GPU,
                   ALL_LAYOUT,
                   phi::TemporalShiftKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
