#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/p_norm_grad_kernel.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/reduce_grad_functions.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"

namespace phi {
template <typename T>
__device__ __forceinline__ int sgn(T val) {
  return (T(0) < val) - (val < T(0));
}

__device__ __forceinline__ int inline_sign(dtype::float16 x) {
  return sgn<dtype::float16>(x);
}
__device__ __forceinline__ int inline_sign(dtype::bfloat16 x) {
  return sgn<dtype::bfloat16>(x);
}
__device__ __forceinline__ int inline_sign(float x) { return sgn<float>(x); }
__device__ __forceinline__ int inline_sign(double x) { return sgn<double>(x); }

__device__ __forceinline__ dtype::float16 inline_abs(dtype::float16 x) {
  return static_cast<dtype::float16>(abs(static_cast<float>(x)));
}

__device__ __forceinline__ dtype::bfloat16 inline_abs(dtype::bfloat16 x) {
  return static_cast<dtype::bfloat16>(abs(static_cast<float>(x)));
}

__device__ __forceinline__ float inline_abs(float x) { return abs(x); }

__device__ __forceinline__ double inline_abs(double x) { return abs(x); }

__device__ __forceinline__ dtype::float16 inline_pow(dtype::float16 base,
                                                     dtype::float16 exponent) {
  return static_cast<dtype::float16>(
      pow(static_cast<float>(base), static_cast<float>(exponent)));
}
__device__ __forceinline__ dtype::bfloat16 inline_pow(
    dtype::bfloat16 base, dtype::bfloat16 exponent) {
  return static_cast<dtype::bfloat16>(
      pow(static_cast<float>(base), static_cast<float>(exponent)));
}
__device__ __forceinline__ float inline_pow(float base, float exponent) {
  return pow(base, exponent);
}
__device__ __forceinline__ double inline_pow(double base, double exponent) {
  return pow(base, exponent);
}

dtype::float16 host_pow(dtype::float16 base, dtype::float16 exponent) {
  return static_cast<dtype::float16>(
      pow(static_cast<float>(base), static_cast<float>(exponent)));
}
dtype::bfloat16 host_pow(dtype::bfloat16 base, dtype::bfloat16 exponent) {
  return static_cast<dtype::bfloat16>(
      pow(static_cast<float>(base), static_cast<float>(exponent)));
}

float host_pow(float base, float exponent) { return pow(base, exponent); }
double host_pow(double base, double exponent) { return pow(base, exponent); }

template <typename T>
struct PNormGradScalarDirectCUDAFunctor {
  const T scalar_;
  const T porder_;

  HOSTDEVICE inline PNormGradScalarDirectCUDAFunctor(const T scalar,
                                                     const T porder)
      : scalar_(scalar), porder_(porder) {}

  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(static_cast<T>(inline_sign(x)) *
                          inline_pow(inline_abs(x), porder_) * scalar_);
  }
};

template <typename T>
struct InfinityNormGradScalarDirectCUDAFunctor {
  const T y_;
  const T dy_;

  HOSTDEVICE inline InfinityNormGradScalarDirectCUDAFunctor(const T y,
                                                            const T dy)
      : y_(y), dy_(dy) {}

  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(dy_ * static_cast<T>(inline_sign(x)) *
                          static_cast<T>((inline_abs(x) == y_)));
  }
};

template <typename T>
struct InfinityNormGradTensorDirectCUDAFunctor {
  HOSTDEVICE inline T operator()(const T x, const T y, const T dy) const {
    return static_cast<T>(dy * static_cast<T>(inline_sign(x)) *
                          static_cast<T>(inline_abs(x) == y));
  }
};

template <typename T>
struct PNormGradTensorDirectCUDAFunctor {
  const T epsilon_;
  const T porder_;

  HOSTDEVICE inline PNormGradTensorDirectCUDAFunctor(const T epsilon,
                                                     const T porder)
      : epsilon_(epsilon), porder_(porder) {}

  HOSTDEVICE inline T operator()(const T x, const T y, const T dy) const {
    return static_cast<T>(
        static_cast<T>(inline_sign(x)) * inline_pow(inline_abs(x), porder_) *
        dy * inline_pow(y + epsilon_, static_cast<T>(-1.0) * porder_));
  }
};

template <typename T, typename Context>
void PNormGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& out,
                     const DenseTensor& out_grad,
                     float porder,
                     int axis,
                     float epsilon,
                     bool keepdim,
                     bool asvector,
                     DenseTensor* x_grad) {
  dev_ctx.template Alloc<T>(x_grad);
  bool reduce_all = (out.numel() == 1);
  if (porder == 0) {
    phi::funcs::SetConstant<Context, T> set_zero;
    set_zero(dev_ctx, x_grad, static_cast<T>(0));
  } else {
    std::vector<DenseTensor*> outputs = {x_grad};
    if (reduce_all) {
      std::vector<const DenseTensor*> inputs = {&x};
      if (porder == INFINITY || porder == -INFINITY) {
        auto functor = InfinityNormGradScalarDirectCUDAFunctor<T>(
            static_cast<T>((*out.data<T>())),
            static_cast<T>((*out_grad.data<T>())));
        funcs::ElementwiseKernel<T>(dev_ctx, inputs, &outputs, functor);
      } else {
        auto functor = PNormGradScalarDirectCUDAFunctor<T>(
            static_cast<T>(static_cast<T>((*out_grad.data<T>())) *
                           host_pow(static_cast<T>((*out.data<T>())) +
                                        static_cast<T>(epsilon),
                                    static_cast<T>(-1.0 * porder))),
            static_cast<T>(porder));
        funcs::ElementwiseKernel<T>(dev_ctx, inputs, &outputs, functor);
      }
    } else {
      std::vector<const DenseTensor*> inputs = {&x, &out, &out_grad};
      if (porder == INFINITY || porder == -INFINITY) {
        auto functor = InfinityNormGradTensorDirectCUDAFunctor<T>();
        funcs::BroadcastKernel<T>(dev_ctx, inputs, &outputs, functor, 0);
      } else {
        auto functor = PNormGradTensorDirectCUDAFunctor<T>(
            static_cast<T>(epsilon), static_cast<T>(porder));
        funcs::BroadcastKernel<T>(dev_ctx, inputs, &outputs, functor, 0);
      }
    }
  }
}
}  // namespace phi
PD_REGISTER_KERNEL(p_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::PNormGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
