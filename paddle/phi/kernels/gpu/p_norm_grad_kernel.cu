#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/p_norm_grad_kernel.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/reduce_grad_functions.h"

namespace phi {

template <typename T>
struct AbsMaxAndMinGradFunctor {
  template <typename Context,
            typename X,
            typename Y,
            typename DX,
            typename DY,
            typename Dim>
  void operator()(const Context& place,
                  X* x,
                  Y* y,
                  DX* dx,
                  DY* dy,
                  const Dim& dim,
                  int size) {
    dx->device(place) = dy->broadcast(dim) * (*x).sign() *
                        ((*x).abs() == y->broadcast(dim)).template cast<T>();
  }
};

template <typename T>
struct PNormGradFunctor {
  HOSTDEVICE explicit inline PNormGradFunctor(float porder, float eps) {
    this->porder = static_cast<T>(porder - 1.);
    this->eps = static_cast<T>(eps);
  }
  template <typename Context,
            typename X,
            typename Y,
            typename DX,
            typename DY,
            typename Dim>
  void operator()(const Context& place,
                  X* x,
                  Y* y,
                  DX* dx,
                  DY* dy,
                  const Dim& dim,
                  int size) {
    dx->device(place) =
        (*x).abs().pow(this->porder) * (*x).sign() * dy->broadcast(dim) *
        (*y + y->constant(eps)).pow(-this->porder).broadcast(dim);
  }
  T porder;
  T eps;
};

template <typename T, typename Context>
void PNormGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& out,
                     const DenseTensor& out_grad,
                     float porder,
                     int axis,
                     float epsilon,
                     bool keepdim,
                     bool asvector,
                     DenseTensor* x_grad) {
  auto* in_x = &x;
  auto* in_norm = &out;
  auto* in_norm_dy = &out_grad;
  auto* out_dx = x_grad;
  dev_ctx.template Alloc<T>(out_dx);

  auto xdim = in_x->dims();
  bool reduce_all = (in_norm->numel() == 1);
  if (axis < 0) axis = xdim.size() + axis;
  const std::vector<int> dims = {axis};

  if (porder == 0) {
    phi::funcs::SetConstant<Context, T> set_zero;
    set_zero(dev_ctx, out_dx, static_cast<T>(0));
  } else if (porder == INFINITY || porder == -INFINITY) {
    AbsMaxAndMinGradFunctor<T> functor;
    funcs::LaunchReduceGradKernel<Context, T, AbsMaxAndMinGradFunctor<T>>(
        dev_ctx, in_x, in_norm, in_norm_dy, out_dx, functor, dims, reduce_all);

  } else {
    auto functor = PNormGradFunctor<T>(porder, epsilon);
    funcs::LaunchReduceGradKernel<Context, T, PNormGradFunctor<T>>(
        dev_ctx, in_x, in_norm, in_norm_dy, out_dx, functor, dims, reduce_all);
  }
}
}  // namespace phi
PD_REGISTER_KERNEL(p_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::PNormGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
