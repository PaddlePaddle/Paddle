#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/psroi_pool_kernel.h"

#include <algorithm>
#include <vector>

#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/common/place.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaximumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaximumNumBlocks);
}

template <typename T>
__global__ void GPUPSROIPoolForward(const int nthreads,
                                    const T* input_data,
                                    const T* input_rois,
                                    const float spatial_scale,
                                    const int input_channels,
                                    const int height,
                                    const int width,
                                    const int output_channels,
                                    const int pooled_height,
                                    const int pooled_width,
                                    const int* rois_batch_id_data,
                                    T* output_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    // The output is in order (n, c, ph, pw)
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % output_channels;
    int n = i / pooled_width / pooled_height / output_channels;

    // set roi_batch_id
    int roi_batch_id = rois_batch_id_data[n];

    // [start, end) interval for spatial sampling
    const T* offset_input_rois = input_rois + n * 4;
    T roi_start_w = static_cast<T>(round(offset_input_rois[0])) * spatial_scale;
    T roi_start_h = static_cast<T>(round(offset_input_rois[1])) * spatial_scale;
    T roi_end_w =
        static_cast<T>(round(offset_input_rois[2]) + 1.) * spatial_scale;
    T roi_end_h =
        static_cast<T>(round(offset_input_rois[3]) + 1.) * spatial_scale;

    // Force too small ROIs to be 1x1
    T roi_height = max(roi_end_h - roi_start_h, (T)0.1);  // avoid 0
    T roi_width = max(roi_end_w - roi_start_w, (T)0.1);

    // Compute w and h at input feature map
    T bin_size_h = roi_height / static_cast<T>(pooled_height);
    T bin_size_w = roi_width / static_cast<T>(pooled_width);

    int hstart = floor(bin_size_h * static_cast<T>(ph) + roi_start_h);
    int wstart = floor(bin_size_w * static_cast<T>(pw) + roi_start_w);
    int hend = ceil(bin_size_h * static_cast<T>(ph + 1) + roi_start_h);
    int wend = ceil(bin_size_w * static_cast<T>(pw + 1) + roi_start_w);

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend = min(max(wend, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int input_channel = (c * pooled_height + ph) * pooled_width + pw;
    const T* offset_input_data =
        input_data +
        (roi_batch_id * input_channels + input_channel) * height * width;
    T outsum = 0;

    for (int ih = hstart; ih < hend; ++ih) {
      for (int iw = wstart; iw < wend; ++iw) {
        int input_index = ih * width + iw;
        outsum += offset_input_data[input_index];
      }
    }

    T bin_area = static_cast<T>((hend - hstart) * (wend - wstart));
    output_data[i] = is_empty ? 0. : outsum / bin_area;
  }
}

template <typename T, typename Context>
void PsroiPoolKernel(const Context& ctx,
                     const DenseTensor& x,
                     const DenseTensor& rois,
                     const paddle::optional<DenseTensor>& rois_num,
                     int pooled_height,
                     int pooled_width,
                     int output_channels,
                     float spatial_scale,
                     DenseTensor* out) {
  auto in_dims = x.dims();
  int batch_size = in_dims[0];
  int input_channels = in_dims[1];
  int height = in_dims[2];
  int width = in_dims[3];

  PADDLE_ENFORCE_EQ(
      input_channels,
      output_channels * pooled_height * pooled_width,
      errors::InvalidArgument(
          "The channels %d of input X should equal the product of "
          "output_channels %d x pooled_height %d x pooled_width %d.",
          input_channels,
          output_channels,
          pooled_height,
          pooled_width));

  int rois_num_t = rois.dims()[0];
  if (rois_num_t == 0) return;
  int rois_batch_size;
  DenseTensor rois_batch_id_list;
  rois_batch_id_list.Resize({rois_num_t});
  int* rois_batch_id_data = ctx.template HostAlloc<int>(&rois_batch_id_list);

  if (rois_num.get_ptr()) {
    rois_batch_size = rois_num->numel();
    auto* rois_num_data = rois_num->data<int>();
    PADDLE_ENFORCE_EQ(rois_batch_size,
                      batch_size,
                      errors::InvalidArgument(
                          "The batch size of input(ROIs) and input(X) must be "
                          "the same but received batch size of input(ROIs) and "
                          "input(X) is %d and %d respectively.",
                          rois_batch_size,
                          batch_size));
    std::vector<int> rois_num_list(rois_batch_size);
    paddle::memory::Copy(CPUPlace(),
                         rois_num_list.data(),
                         ctx.GetPlace(),
                         rois_num_data,
                         sizeof(int) * rois_batch_size,
                         0);
    int rois_num_count = 0;
    for (int i = 0; i < rois_batch_size; ++i) {
      rois_num_count += rois_num_list[i];
    }
    PADDLE_ENFORCE_EQ(
        rois_num_count,
        rois_num_t,
        errors::InvalidArgument(
            "the rois_num from input and RoisNum must be the same"));
    int start = 0;
    for (int n = 0; n < rois_batch_size; ++n) {
      for (int i = start; i < start + rois_num_list[n]; ++i) {
        rois_batch_id_data[i] = n;
      }
      start += rois_num_list[n];
    }
  } else {
    auto rois_lod = rois.lod().back();
    rois_batch_size = rois_lod.size() - 1;
    PADDLE_ENFORCE_EQ(rois_batch_size,
                      batch_size,
                      errors::InvalidArgument(
                          "The batch size of input(ROIs) and input(X) must be "
                          "the same but received batch size of input(ROIs) and "
                          "input(X) is %d and %d respectively.",
                          rois_batch_size,
                          batch_size));
    int rois_num_with_lod = rois_lod[rois_batch_size];
    PADDLE_ENFORCE_EQ(rois_num_t,
                      rois_num_with_lod,
                      errors::InvalidArgument(
                          "The number of rois from input(ROIs) and its LOD "
                          "must be the same. Received rois %d of input(ROIs) "
                          "but the number of rois %d from its LOD is %d",
                          rois_num,
                          rois_num_with_lod));

    // set rois batch id
    for (int n = 0; n < rois_batch_size; ++n) {
      for (size_t i = rois_lod[n]; i < rois_lod[n + 1]; ++i) {
        rois_batch_id_data[i] = n;
      }
    }
  }
  DenseTensor rois_batch_id_list_gpu;
  Copy(ctx, rois_batch_id_list, ctx.GetPlace(), false, &rois_batch_id_list_gpu);

  int output_size = out->numel();
  int blocks = NumBlocks(output_size);
  int threads = kNumCUDAThreads;

  // call cuda kernel function
  GPUPSROIPoolForward<T>
      <<<blocks, threads, 0, ctx.stream()>>>(output_size,
                                             x.data<T>(),
                                             rois.data<T>(),
                                             spatial_scale,
                                             input_channels,
                                             height,
                                             width,
                                             output_channels,
                                             pooled_height,
                                             pooled_width,
                                             rois_batch_id_list_gpu.data<int>(),
                                             ctx.template Alloc<T>(out));
}

}  // namespace phi

PD_REGISTER_KERNEL(
    psroi_pool, GPU, ALL_LAYOUT, phi::PsroiPoolKernel, float, double) {
  kernel->InputAt(2).SetDataType(
      paddle::experimental::CppTypeToDataType<int>::Type());
}
