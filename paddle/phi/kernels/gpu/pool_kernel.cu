// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/pool_kernel.h"

#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/pool_kernel_impl.h"

PD_REGISTER_KERNEL(pool2d,
                   GPU,
                   ALL_LAYOUT,
                   phi::Pool2dKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(max_pool2d_with_index,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaxPool2dWithIndexKernel,
                   float,
                   double) {
  kernel->OutputAt(1).SetDataType(
      paddle::experimental::CppTypeToDataType<int>::Type());
}

PD_REGISTER_KERNEL(pool3d,
                   GPU,
                   ALL_LAYOUT,
                   phi::Pool3dKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
PD_REGISTER_KERNEL(max_pool3d_with_index,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaxPool3dWithIndexKernel,
                   float,
                   double) {
  kernel->OutputAt(1).SetDataType(
      paddle::experimental::CppTypeToDataType<int>::Type());
}
