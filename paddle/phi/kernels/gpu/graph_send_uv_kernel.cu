#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/graph_send_uv_kernel.h"
#include "paddle/phi/kernels/gpu/graph_send_ue_recv_funcs.h"
#include "paddle/phi/kernels/impl/graph_message_passing_impl.h"

#include <thrust/device_vector.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/hostdevice.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"

namespace phi {

template <typename T, typename IndexT, typename ComputeFunctor>
__global__ void GraphSendUVCUDAKernel(const T* x_data,
                                      const T* y_data,
                                      const IndexT* src_indices,
                                      const IndexT* dst_indices,
                                      const int64_t* xbcast_off,
                                      const int64_t* ybcast_off,
                                      T* output,
                                      int64_t index_size,
                                      int64_t x_len,
                                      int64_t y_len,
                                      int64_t out_len,
                                      bool use_bcast,
                                      ComputeFunctor cfunctor) {
  IndexT ty = blockIdx.y * blockDim.y + threadIdx.y;
  const IndexT stride_y = blockDim.y * gridDim.y;

  while (ty < index_size) {
    IndexT src = src_indices[ty];
    IndexT dst = dst_indices[ty];
    int64_t tx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t stride_x = blockDim.x * gridDim.x;

    const T* x_off = x_data + src * x_len;
    const T* y_off = y_data + dst * y_len;
    T* out_off = output + ty * out_len;
    while (tx < out_len) {
      int64_t x_add = use_bcast ? xbcast_off[tx] : tx;
      int64_t y_add = use_bcast ? ybcast_off[tx] : tx;
      T val = cfunctor(x_off[x_add], y_off[y_add]);
      out_off[tx] = val;
      tx += stride_x;
    }
    ty += stride_y;
  }
}

template <typename Context, typename T, typename IndexT>
void GraphSendUVOpCUDAKernelLaunchHelper(const Context& ctx,
                                         const DenseTensor& x,
                                         const DenseTensor& y,
                                         const DenseTensor& src_index,
                                         const DenseTensor& dst_index,
                                         const std::string& message_op,
                                         DenseTensor* out) {
  const int64_t& index_size = src_index.dims()[0];
  PADDLE_ENFORCE_GT(
      index_size,
      0,
      errors::InvalidArgument("The first dimension of src_index or dst_index "
                              "shoule be greater than 0, but received %d.",
                              index_size));

  auto out_dims = out->dims();
  int64_t memset_size = 1;
  for (int i = 0; i < out_dims.size(); i++) {
    memset_size *= out_dims[i];
  }
  ctx.template Alloc<T>(out);
  T* out_data = out->data<T>();

  const auto& bcast_info = phi::CalcBCastInfo(x.dims(), y.dims());
  const T* x_data = x.data<T>();
  const T* y_data = y.data<T>();
  const IndexT* s_index = src_index.data<IndexT>();
  const IndexT* d_index = dst_index.data<IndexT>();

  thrust::device_vector<int64_t> x_bcastoff, y_bcastoff;
  if (bcast_info.use_bcast) {
    CopyBCastOff(bcast_info, x_bcastoff, y_bcastoff);
  }

  int64_t out_len = bcast_info.out_len;
  const int ntx = FindNumThreads(out_len, ctx.GetMaxThreadsPerBlock());
  const int nty = ctx.GetMaxThreadsPerBlock() / ntx;
  const int nbx = (out_len + ntx - 1) / ntx;
  const int nby = FindNumBlocks('y', (index_size + nty - 1) / nty);
  const dim3 grid(nbx, nby);
  const dim3 block(ntx, nty);
  if (message_op == "ADD") {
    funcs::AddFunctor<T> add_functor;
    GraphSendUVCUDAKernel<T, IndexT, funcs::AddFunctor<T>>
        <<<grid, block, 0, ctx.stream()>>>(
            x_data,
            y_data,
            s_index,
            d_index,
            thrust::raw_pointer_cast(x_bcastoff.data()),
            thrust::raw_pointer_cast(y_bcastoff.data()),
            out_data,
            index_size,
            bcast_info.l_len,
            bcast_info.r_len,
            out_len,
            bcast_info.use_bcast,
            add_functor);
  } else if (message_op == "MUL") {
    funcs::MultiplyFunctor<T> mul_functor;
    GraphSendUVCUDAKernel<T, IndexT, funcs::MultiplyFunctor<T>>
        <<<grid, block, 0, ctx.stream()>>>(
            x_data,
            y_data,
            s_index,
            d_index,
            thrust::raw_pointer_cast(x_bcastoff.data()),
            thrust::raw_pointer_cast(y_bcastoff.data()),
            out_data,
            index_size,
            bcast_info.l_len,
            bcast_info.r_len,
            out_len,
            bcast_info.use_bcast,
            mul_functor);
  }
}

template <typename T, typename Context>
void GraphSendUVKernel(const Context& ctx,
                       const DenseTensor& x,
                       const DenseTensor& y,
                       const DenseTensor& src_index,
                       const DenseTensor& dst_index,
                       const std::string& message_op,
                       DenseTensor* out) {
  auto index_type = src_index.dtype();
  if (index_type == phi::DataType::INT32) {
    GraphSendUVOpCUDAKernelLaunchHelper<Context, T, int32_t>(
        ctx, x, y, src_index, dst_index, message_op, out);
  } else if (index_type == phi::DataType::INT64) {
    GraphSendUVOpCUDAKernelLaunchHelper<Context, T, int64_t>(
        ctx, x, y, src_index, dst_index, message_op, out);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(graph_send_uv,
                   GPU,
                   ALL_LAYOUT,
                   phi::GraphSendUVKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
