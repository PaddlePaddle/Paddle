// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/nll_loss_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/nll_loss.h"

namespace phi {
template <typename T, typename Context>
void NllLossGradKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& labels,
                       const paddle::optional<DenseTensor>& weight,
                       const DenseTensor& total_weight,
                       const DenseTensor& dout,
                       int64_t ignore_index,
                       const std::string& reduction,
                       DenseTensor* dx) {
  auto dx_data = dev_ctx.template Alloc<T>(dx);
  auto dout_data = dout.data<T>();
  auto label_data = labels.data<int64_t>();
  auto weight_data = weight.get_ptr() ? weight.get_ptr()->data<T>() : nullptr;
  auto total_weight_data = total_weight.data<T>();
#ifdef PADDLE_WITH_HIP
  hipMemset(dx_data, 0, dx->numel() * sizeof(T));
#else
  hipMemset(dx_data, 0, dx->numel() * sizeof(T));
#endif

  int64_t size_average = (int64_t)(reduction == "mean");
  auto x_dims = x.dims();
  auto batch_size = x_dims[0];
  auto n_classes = x_dims[1];

  if (x_dims.size() == 2) {
    int blocks = NumBlocks(batch_size);
    int threads = kNumCUDAThreads;
    if (reduction == "none") {
      GPUNLLLossBackward1D_no_reduce<T>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(dx_data,
                                                     label_data,
                                                     weight_data,
                                                     dout_data,
                                                     batch_size,
                                                     n_classes,
                                                     ignore_index);
    } else {
      GPUNLLLossBackward1D_with_reduce<T>
          <<<1, NTHREADS, 0, dev_ctx.stream()>>>(dx_data,
                                                 total_weight_data,
                                                 label_data,
                                                 weight_data,
                                                 dout_data,
                                                 batch_size,
                                                 n_classes,
                                                 size_average,
                                                 ignore_index);
    }
  } else if (x_dims.size() == 4) {
    const auto in_dim2 = x_dims[2];
    const auto in_dim3 = x_dims[3];
    const auto map_size = in_dim2 * in_dim3;
    const auto out_numel = batch_size * in_dim2 * in_dim3;

    int blocks = NumBlocks(out_numel);
    int threads = kNumCUDAThreads;
    if (reduction == "none") {
      GPUNLLLossBackward2D_no_reduce<T>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(dx_data,
                                                     label_data,
                                                     weight_data,
                                                     dout_data,
                                                     batch_size,
                                                     n_classes,
                                                     in_dim2,
                                                     in_dim3,
                                                     ignore_index);
    } else {
      int blocks_per_sample = NumBlocks(map_size) / 128;
      blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
      int total_blocks = blocks_per_sample * batch_size;
      GPUNLLLossBackward2D_with_reduce<T>
          <<<total_blocks, threads, 0, dev_ctx.stream()>>>(dx_data,
                                                           total_weight_data,
                                                           label_data,
                                                           weight_data,
                                                           dout_data,
                                                           batch_size,
                                                           n_classes,
                                                           map_size,
                                                           blocks_per_sample,
                                                           size_average,
                                                           ignore_index);
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(
    nll_loss_grad, GPU, ALL_LAYOUT, phi::NllLossGradKernel, float, double) {}
