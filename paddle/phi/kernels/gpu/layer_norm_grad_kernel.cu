// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/layer_norm_grad_kernel.h"

#include "paddle/fluid/operators/layer_norm_kernel.cu.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/layer_norm_util.h"

namespace phi {

template <typename T, typename Context>
void LayerNormGradKernel(const Context &dev_ctx,
                         const DenseTensor &x,
                         const paddle::optional<DenseTensor> &scale_opt,
                         const paddle::optional<DenseTensor> &bias_opt,
                         const DenseTensor &mean,
                         const DenseTensor &variance,
                         const DenseTensor &out_grad,
                         float epsilon,
                         int begin_norm_axis,
                         bool is_test,
                         DenseTensor *x_grad,
                         DenseTensor *scale_grad,
                         DenseTensor *bias_grad) {
  using U = paddle::operators::LayerNormParamType<T>;
  // d_x, d_scale, d_bias may be nullptr
  auto *d_x = x_grad;
  auto *d_scale = scale_grad;
  auto *d_bias = bias_grad;

  auto *scale = scale_opt.get_ptr();
  auto *bias = bias_opt.get_ptr();
  auto *d_y = &out_grad;

  const auto &x_dims = x.dims();
  auto matrix_dim = phi::flatten_to_2d(x_dims, begin_norm_axis);
  int64_t batch_size = static_cast<int64_t>(matrix_dim[0]);
  int64_t feature_size = static_cast<int64_t>(matrix_dim[1]);

  auto *x_data = x.data<T>();
  auto *d_y_data = d_y->data<T>();

  auto *mean_data = mean.data<U>();
  auto *var_data = variance.data<U>();

  auto *d_x_data = (d_x == nullptr ? nullptr : dev_ctx.template Alloc<T>(d_x));

  auto x_dtype = x.dtype();

  phi::DataType scale_bias_dtype;
  if (scale != nullptr) {
    scale_bias_dtype = scale->dtype();
  } else {
    // FIXME(zengjinle): do not find a better way to get the right
    // data type of the d_scale and d_bias if scale == nullptr.
    if (bias != nullptr) {
      scale_bias_dtype = bias->dtype();
    } else {
      scale_bias_dtype = x_dtype;
    }
  }

#define PADDLE_LAUNCH_LAYERNORM_BWD(ScaleBiasT, IsScaleBiasSameDTypeWithX)  \
  do {                                                                      \
    auto *scale_data =                                                      \
        (scale == nullptr ? nullptr : scale->data<ScaleBiasT>());           \
    auto *d_scale_data =                                                    \
        (d_scale == nullptr ? nullptr                                       \
                            : dev_ctx.template Alloc<ScaleBiasT>(d_scale)); \
    auto *d_bias_data =                                                     \
        (d_bias == nullptr ? nullptr                                        \
                           : dev_ctx.template Alloc<ScaleBiasT>(d_bias));   \
    auto *d_x_data =                                                        \
        (d_x == nullptr ? nullptr : dev_ctx.template Alloc<T>(d_x));        \
    paddle::operators::LayerNormBackward<T, U, IsScaleBiasSameDTypeWithX>(  \
        x_data,                                                             \
        d_y_data,                                                           \
        scale_data,                                                         \
        mean_data,                                                          \
        var_data,                                                           \
        d_x_data,                                                           \
        d_scale_data,                                                       \
        d_bias_data,                                                        \
        epsilon,                                                            \
        batch_size,                                                         \
        feature_size,                                                       \
        dev_ctx);                                                           \
  } while (0)

  if (scale_bias_dtype == x_dtype) {
    PADDLE_LAUNCH_LAYERNORM_BWD(T, true);
  } else {
    PADDLE_LAUNCH_LAYERNORM_BWD(U, false);
  }

#undef PADDLE_LAUNCH_LAYERNORM_BWD
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(layer_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::LayerNormGradKernel,
                   float,
                   phi::dtype::float16) {}
#elif CUDNN_VERSION_MIN(8, 1, 0)
PD_REGISTER_KERNEL(layer_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::LayerNormGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#else
PD_REGISTER_KERNEL(layer_norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::LayerNormGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#endif
