#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "paddle/phi/kernels/weight_only_matmul_kernel.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/datatype_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/weight_only_gemv.h"
#if defined(PADDLE_WITH_CUTLASS)
#include "paddle/phi/kernels/fusion/cutlass/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"
#endif

namespace phi {

template <typename T, typename Context>
void WeightOnlyMatmulKernel(const Context& dev_ctx,
                            const DenseTensor& x,
                            const DenseTensor& weight,
                            const paddle::optional<DenseTensor>& bias,
                            const DenseTensor& weight_scale,
                            const std::string& quant_method,
                            DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  const T* x_data = x.data<T>();
  const int8_t* weight_data = weight.data<int8_t>();
  const T* bias_data = bias ? bias.get().data<T>() : nullptr;
  const float* weight_scale_data = weight_scale.data<float>();
  T* out_data = out->data<T>();
  const auto x_dims = x.dims();
  const auto w_dims = weight.dims();
  int n = weight_scale.dims()[0];
  int k = w_dims[1];
  int m = x.numel() / k;

  // m > 1: run gemm
  if (m > 1 || quant_method == "weight_only_int4") {
#if defined(PADDLE_WITH_CUTLASS)
    if (quant_method == "weight_only_int8") {
      auto mixed_gemm_runner =
          CutlassFpAIntBGemmRunner<typename PDDataTypeTraits<T>::DataType,
                                   uint8_t>();
      int mixgemm_max_size = std::max(n, k);
      DenseTensor mixgemm_workspace;
      int64_t mixgemm_workspace_size_bytes = mixed_gemm_runner.getWorkspaceSize(
          m, mixgemm_max_size, mixgemm_max_size);

      mixgemm_workspace.Resize({mixgemm_workspace_size_bytes});
      dev_ctx.template Alloc<uint8_t>(&mixgemm_workspace);
      char* mixgemm_workspace_data =
          reinterpret_cast<char*>(mixgemm_workspace.data<uint8_t>());
      if (bias_data) {
        mixed_gemm_runner.gemm_bias_act(
            reinterpret_cast<const typename PDDataTypeTraits<T>::DataType*>(
                x_data),
            reinterpret_cast<const uint8_t*>(weight_data),
            weight_scale_data,
            reinterpret_cast<const typename PDDataTypeTraits<T>::DataType*>(
                bias_data),
            reinterpret_cast<typename PDDataTypeTraits<T>::DataType*>(out_data),
            m,
            n,
            k,
            "none",
            mixgemm_workspace_data,
            mixgemm_workspace_size_bytes,
            dev_ctx.stream());
      } else {
        mixed_gemm_runner.gemm(
            reinterpret_cast<const typename PDDataTypeTraits<T>::DataType*>(
                x_data),
            reinterpret_cast<const uint8_t*>(weight_data),
            weight_scale_data,
            reinterpret_cast<typename PDDataTypeTraits<T>::DataType*>(out_data),
            m,
            n,
            k,
            mixgemm_workspace_data,
            mixgemm_workspace_size_bytes,
            dev_ctx.stream());
      }
    } else {
      auto mixed_gemm_runner =
          CutlassFpAIntBGemmRunner<typename PDDataTypeTraits<T>::DataType,
                                   cutlass::uint4b_t>();
      int mixgemm_max_size = std::max(n, k);
      DenseTensor mixgemm_workspace;
      int64_t mixgemm_workspace_size_bytes = mixed_gemm_runner.getWorkspaceSize(
          m, mixgemm_max_size, mixgemm_max_size);

      mixgemm_workspace.Resize({mixgemm_workspace_size_bytes});
      dev_ctx.template Alloc<uint8_t>(&mixgemm_workspace);
      char* mixgemm_workspace_data =
          reinterpret_cast<char*>(mixgemm_workspace.data<uint8_t>());
      if (bias_data) {
        mixed_gemm_runner.gemm_bias_act(
            reinterpret_cast<const typename PDDataTypeTraits<T>::DataType*>(
                x_data),
            reinterpret_cast<const cutlass::uint4b_t*>(weight_data),
            weight_scale_data,
            reinterpret_cast<const typename PDDataTypeTraits<T>::DataType*>(
                bias_data),
            reinterpret_cast<typename PDDataTypeTraits<T>::DataType*>(out_data),
            m,
            n,
            k,
            "none",
            mixgemm_workspace_data,
            mixgemm_workspace_size_bytes,
            dev_ctx.stream());
      } else {
        mixed_gemm_runner.gemm(
            reinterpret_cast<const typename PDDataTypeTraits<T>::DataType*>(
                x_data),
            reinterpret_cast<const cutlass::uint4b_t*>(weight_data),
            weight_scale_data,
            reinterpret_cast<typename PDDataTypeTraits<T>::DataType*>(out_data),
            m,
            n,
            k,
            mixgemm_workspace_data,
            mixgemm_workspace_size_bytes,
            dev_ctx.stream());
      }
    }
#else
    LOG(ERROR) << "Please compile with cutlass to EnableUseCutlass()";
#endif
  } else {  // m == 1: gemv
    if (quant_method == "weight_only_int8") {
      GemvWeightonlyInt8Wrapper<T, Context>(dev_ctx,
                                            x_data,
                                            weight_data,
                                            bias_data,
                                            weight_scale_data,
                                            n,
                                            k,
                                            "None",
                                            out->data<T>());
    }  // TODO(lizhenyun) support weight_only_gemv_int4.
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(weight_only_matmul,
                   GPU,
                   ALL_LAYOUT,
                   phi::WeightOnlyMatmulKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
