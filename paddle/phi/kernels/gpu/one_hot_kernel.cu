#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/one_hot_kernel.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename InT, typename OutT>
__global__ void FillOutputKernel(const InT* p_in_data,
                                 OutT* p_out_data,
                                 const int64_t numel,
                                 const int depth) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < numel && p_in_data[idx] >= 0 && p_in_data[idx] < depth) {
    *(p_out_data + (idx * depth) + p_in_data[idx]) = 1.0;
  }
}

template <typename DeviceContext, typename InT>
struct OneHotV2OpCUDAFunctor {
  const DenseTensor* in_;
  DenseTensor* out_;
  const DeviceContext& ctx_;
  int depth_;

  OneHotV2OpCUDAFunctor(const DenseTensor* in,
                        DenseTensor* out,
                        int depth,
                        const DeviceContext& ctx)
      : in_(in), out_(out), depth_(depth), ctx_(ctx) {}

  template <typename OutT>
  void apply() const {
    auto* p_in_data = in_->data<InT>();
    auto numel = in_->numel();
    auto* p_out_data = ctx_.template Alloc<OutT>(out_);
    auto stream = ctx_.stream();
    funcs::set_constant(ctx_, out_, 0.0);

    FillOutputKernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                           PADDLE_CUDA_NUM_THREADS,
                       PADDLE_CUDA_NUM_THREADS,
                       0,
                       stream>>>(p_in_data, p_out_data, numel, depth_);
  }
};

template <typename T, typename Context>
void OneHotRawKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     int32_t depth,
                     int dtype,
                     bool allow_out_of_range,
                     DenseTensor* out) {
  auto out_dims = out->dims();
  if (out_dims[out_dims.size() - 1] == -1) {
    out_dims[out_dims.size() - 1] = depth;
    out->Resize(out_dims);
  }

  paddle::framework::VisitDataType(
      static_cast<paddle::framework::proto::VarType::Type>(dtype),
      OneHotV2OpCUDAFunctor<Context, T>(&x, out, depth, dev_ctx));
}

template <typename T, typename Context>
void OneHotKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  int32_t num_classes,
                  DenseTensor* out) {
  auto out_dims = out->dims();
  if (out_dims[out_dims.size() - 1] == -1) {
    out_dims[out_dims.size() - 1] = num_classes;
    out->Resize(out_dims);
  }

  OneHotV2OpCUDAFunctor<Context, T> one_hot_func(&x, out, num_classes, dev_ctx);
  one_hot_func.template apply<float>();
}

}  // namespace phi

PD_REGISTER_KERNEL(
    one_hot_raw, GPU, ALL_LAYOUT, phi::OneHotRawKernel, int, int64_t) {}

PD_REGISTER_KERNEL(one_hot, GPU, ALL_LAYOUT, phi::OneHotKernel, int, int64_t) {}
