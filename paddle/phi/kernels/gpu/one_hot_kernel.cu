#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/one_hot_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename InT, typename OutT>
__global__ void FillOutputKernel(const InT* p_in_data,
                                 OutT* p_out_data,
                                 const int64_t numel,
                                 const int depth) {
  CUDA_KERNEL_LOOP_TYPE(idx, numel, int64_t) {
    PADDLE_ENFORCE(p_in_data[idx] >= 0 && p_in_data[idx] < depth,
                   "Illegal index value, Input(input) value should be "
                   "greater than or equal to 0, and less than depth [%d], "
                   "but received [%lld].",
                   depth,
                   p_in_data[idx]);

    *(p_out_data + (idx * depth) + p_in_data[idx]) = 1.0;
  }
}

template <typename DeviceContext, typename InT>
struct OneHotV2OpCUDAFunctor {
  const DenseTensor* in_;
  DenseTensor* out_;
  const DeviceContext& ctx_;
  int depth_;

  OneHotV2OpCUDAFunctor(const DenseTensor* in,
                        DenseTensor* out,
                        int depth,
                        const DeviceContext& ctx)
      : in_(in), out_(out), depth_(depth), ctx_(ctx) {}

  template <typename OutT>
  void apply() const {
    auto* p_in_data = in_->data<InT>();
    auto numel = in_->numel();
    auto* p_out_data = ctx_.template Alloc<OutT>(out_);
    auto stream = ctx_.stream();
    funcs::set_constant(ctx_, out_, 0.0);

    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(ctx_, numel);

    FillOutputKernel<<<config.block_per_grid,
                       config.thread_per_block,
                       0,
                       stream>>>(p_in_data, p_out_data, numel, depth_);
  }
};

template <typename T, typename Context>
void OneHotRawKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const Scalar& depth,
                     DataType dtype,
                     bool allow_out_of_range,
                     DenseTensor* out) {
  auto depth_v = depth.to<int>();
  auto out_dims = out->dims();
  if (out_dims[out_dims.size() - 1] == -1) {
    out_dims[out_dims.size() - 1] = depth_v;
    out->Resize(out_dims);
  }

  phi::VisitDataType(
      dtype, OneHotV2OpCUDAFunctor<Context, T>(&x, out, depth_v, dev_ctx));
}

}  // namespace phi

PD_REGISTER_KERNEL(
    one_hot_raw, GPU, ALL_LAYOUT, phi::OneHotRawKernel, int, int64_t) {}
