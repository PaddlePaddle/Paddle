#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/adamax_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
namespace phi {
template <typename T, typename MT>
__global__ void AdamaxGPUKernel(const T* param,
                                const T* grad,
                                const MT* learning_rate,
                                const MT* moment,
                                const MT* inf_norm,
                                const MT* beta1_pow,
                                const MT* master_param,
                                MT d_beta1,
                                MT d_beta2,
                                MT d_epsilon,
                                int num,
                                T* param_out,
                                MT* moment_out,
                                MT* inf_norm_out,
                                MT* master_param_out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  MT lr = static_cast<MT>(learning_rate[0]);
  MT d_pow = static_cast<MT>(beta1_pow[0]);
  MT one = static_cast<MT>(1.0f);
  auto l_r = lr / (one - d_pow);

  for (int index = idx; index < num; index += gridDim.x * blockDim.x) {
    // load and cast input to MT
    MT d_param =
        master_param ? master_param[index] : static_cast<MT>(param[index]);
    MT d_grad = static_cast<MT>(grad[index]);
    MT d_moment = static_cast<MT>(moment[index]);
    MT d_inf = static_cast<MT>(inf_norm[index]);
    // compute
    auto mom_out = d_beta1 * d_moment + (one - d_beta1) * d_grad;
    auto norm_out = std::max(std::abs(d_grad), d_beta2 * d_inf + d_epsilon);
    auto out_data = d_param - l_r * (mom_out / norm_out);
    // store
    param_out[index] = static_cast<T>(out_data);
    moment_out[index] = static_cast<T>(mom_out);
    inf_norm_out[index] = static_cast<T>(norm_out);

    if (master_param_out) {
      master_param_out[index] = out_data;
    }
  }
}

template <typename T, typename Context>
void AdamaxKernel(const Context& dev_ctx,
                  const DenseTensor& param,
                  const DenseTensor& grad,
                  const DenseTensor& learning_rate,
                  const DenseTensor& moment,
                  const DenseTensor& inf_norm,
                  const DenseTensor& beta1_pow,
                  const paddle::optional<DenseTensor>& master_param,
                  float beta1,
                  float beta2,
                  float epsilon,
                  bool multi_precision,
                  DenseTensor* param_out,
                  DenseTensor* moment_out,
                  DenseTensor* inf_norm_out,
                  DenseTensor* master_param_outs) {
  using MPDType = typename phi::dtype::template MPTypeTrait<T>::Type;
  T* param_out_data = dev_ctx.template Alloc<T>(param_out);
  MPDType* moment_out_data = dev_ctx.template Alloc<MPDType>(moment_out);
  MPDType* inf_norm_out_data = dev_ctx.template Alloc<MPDType>(inf_norm_out);
  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision ? dev_ctx.template Alloc<MPDType>(master_param_outs)
                      : nullptr;
  PADDLE_ENFORCE_EQ(
      beta1_pow.numel(),
      1,
      errors::InvalidArgument("beta1 pow's size should be 1, but received "
                              "value is:%d.",
                              beta1_pow.numel()));

  MPDType beta1_ = static_cast<MPDType>(beta1);
  MPDType beta2_ = static_cast<MPDType>(beta2);
  MPDType epsilon_ = static_cast<MPDType>(epsilon);

  int numel = param.numel();
  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, 1);
  int grid = config.block_per_grid.x;
  int block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  AdamaxGPUKernel<T, MPDType>
      <<<block, grid, 0, stream>>>(param.data<T>(),
                                   grad.data<T>(),
                                   learning_rate.data<MPDType>(),
                                   moment.data<MPDType>(),
                                   inf_norm.data<MPDType>(),
                                   beta1_pow.data<MPDType>(),
                                   master_in_data,
                                   beta1_,
                                   beta2_,
                                   epsilon_,
                                   numel,
                                   param_out_data,
                                   moment_out_data,
                                   inf_norm_out_data,
                                   master_out_data);
}
}  // namespace phi
PD_REGISTER_KERNEL(adamax,
                   GPU,
                   ALL_LAYOUT,
                   phi::AdamaxKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  if (kernel_key.dtype() == phi::DataType::FLOAT16) {
    kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(3).SetDataType(phi::DataType::FLOAT32);
  }
}
