// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/reduce_mean_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"

namespace phi {

template <typename T, typename Context>
void ReduceMeanGradKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& out_grad,
                          const IntArray& dims,
                          bool keep_dim,
                          bool reduce_all,
                          DenseTensor* x_grad) {
<<<<<<< HEAD
  int dim_size = x.dims().size();
  std::vector<int> reduce_dims =
      funcs::details::GetReduceDim(dims, dim_size, reduce_all);
  int reduce_num = 1;
  for (auto i : reduce_dims) {
    reduce_num *= (x.dims())[i];
  }
  using MPType = typename kps::details::MPTypeTrait<T>::Type;
  ReduceGradKernel<T, T, Context, kps::DivideFunctor<T, MPType>>(
      dev_ctx,
      x,
      out_grad,
      dims,
      keep_dim,
      reduce_all,
      x_grad,
      kps::DivideFunctor<T, MPType>(reduce_num));
=======
  reduce_all = recompute_reduce_all(x, dims, reduce_all);
  // get reduce_dim and reduce_num for reduce_mean_grad
  int dim_size = x.dims().size();
  std::vector<int> reduce_dims =
      funcs::details::GetReduceDim(dims.GetData(), dim_size, reduce_all);

  auto update_dims = vectorize(x.dims());
  int reduce_num = 1;
  for (auto i : reduce_dims) {
    reduce_num *= (x.dims())[i];
    update_dims[i] = 1;
  }

  // make new tensor
  DenseTensor new_out_grad(out_grad.dtype());
  new_out_grad.ShareDataWith(out_grad);
  new_out_grad.Resize(phi::make_ddim(update_dims));

  // call BroadcastKernel
  dev_ctx.Alloc(x_grad, x.dtype());
  std::vector<const DenseTensor*> inputs = {&new_out_grad};
  std::vector<DenseTensor*> outputs = {x_grad};

  using MPType = typename kps::details::MPTypeTrait<T>::Type;
  funcs::BroadcastKernel<phi::ElementwiseType::kUnary, T, T>(
      dev_ctx, inputs, &outputs, 0, kps::DivideFunctor<T, MPType>(reduce_num));
>>>>>>> d828ca460a89c2ce88be15bb5cdb76c676decf91
}

}  // namespace phi

PD_REGISTER_KERNEL(mean_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ReduceMeanGradKernel,
                   bool,
                   float,
                   double,
                   phi::dtype::float16) {}
