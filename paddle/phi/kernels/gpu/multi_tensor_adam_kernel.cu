#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/multi_tensor_adam_kernel.h"
#include <assert.h>
#include <cstdlib>
#include <vector>
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/scalar.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/adam_functors.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/funcs/selected_rows_functor.h"
#include "paddle/phi/kernels/gpu/multi_tensor_apply_kernel.cuh"

namespace phi {

#define BLOCK_SIZE 512
#define ILP 4

using MATH_T = float;

template <typename T, typename MT, int N>
struct AdamFunctor {
    __device__ __forceinline__ void operator()(int chunk_size,
                                             TensorListMetadata<N> tl,
                                             MT beta1,
                                             MT beta2,
                                             const MT* beta1_pow_,
                                             const MT* beta2_pow_,
                                             MT epsilon,
                                             const MT* learning_rate,
                                             adamMode_t mode,
                                             bool multi_precision,
                                             MT decay) {
    MT lr = *learning_rate;
    MT beta1_pow = *beta1_pow_;
    MT beta2_pow = *beta2_pow_;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    int chunk_idx = tl.block_to_chunk[blockIdx.x] + tl.start_chunk_this_tensor;

    int n = tl.sizes[tensor_loc];
    T* g = static_cast<T*>(tl.addresses[0][tensor_loc]);
    g += chunk_idx*chunk_size;
    MT* mp;
    T* p;
    p = static_cast<T*>(tl.addresses[1][tensor_loc]);
    p += chunk_idx*chunk_size;
    MT* m = static_cast<MT*>(tl.addresses[2][tensor_loc]);
    m += chunk_idx*chunk_size;
    MT* v = static_cast<MT*>(tl.addresses[3][tensor_loc]);
    v += chunk_idx*chunk_size;
    mp = static_cast<MT*>(tl.addresses[4][tensor_loc]);
    mp += chunk_idx*chunk_size;
    n -= chunk_idx*chunk_size;
    for (int i_start = 0; i_start < n && i_start < chunk_size;
         i_start += blockDim.x * ILP) {
      MT r_g[ILP];
      MT r_p[ILP];
      MT r_m[ILP];
      MT r_v[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = static_cast<MT>(g[i]);
          r_p[ii] = multi_precision ? mp[i] : static_cast<MT>(p[i]);
          r_m[ii] = static_cast<MT>(m[i]);
          r_v[ii] = static_cast<MT>(v[i]);
        } else {
          r_g[ii] = MT(0);
          r_p[ii] = MT(0);
          r_m[ii] = MT(0);
          r_v[ii] = MT(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        MT p = r_p[ii];
        MT g = r_g[ii];
        MT m = r_m[ii];
        MT v = r_v[ii];
        if (mode == ADAM_MODE_0) {
          m = beta1 * m + (static_cast<MT>(1.0) - beta1) * g;
          v = beta2 * v + (static_cast<MT>(1.0) - beta2) * g * g;
          r_m[ii] = m;
          r_v[ii] = v;
          MT denom =
              (sqrt(v) / sqrt(static_cast<MT>(1.0) - beta2_pow)) + epsilon;
          p += (m / denom) * (-(lr / (static_cast<MT>(1.0) - beta1_pow)));
          r_p[ii] = p;
        } else {  // weight decay
          p *= (static_cast<MT>(1.0) - lr * decay);
          m = beta1 * m + (static_cast<MT>(1.0) - beta1) * g;
          v = beta2 * v + (static_cast<MT>(1.0) - beta2) * g * g;
          r_m[ii] = m;
          r_v[ii] = v;
          MT denom =
              (sqrt(v) / sqrt(static_cast<MT>(1.0) - beta2_pow)) + epsilon;
          p += (m / denom) * (-(lr / (static_cast<MT>(1.0) - beta1_pow)));
          r_p[ii] = p;
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = static_cast<T>(r_p[ii]);
          m[i] = r_m[ii];
          v[i] = r_v[ii];
          if (multi_precision) {
            mp[i] = r_p[ii];
          }
        }
      }
    }
  }
};

template <typename T>
__global__ void UpdateBetaPow(T beta1,
                              T beta2,
                              const T* beta1_pow_,
                              const T* beta2_pow_,
                              T* beta1_pow_out,
                              T* beta2_pow_out) {
  *beta1_pow_out = beta1 * beta1_pow_[0];
  *beta2_pow_out = beta2 * beta2_pow_[0];
}

template <typename T, typename Context>
void MultiTensorAdamKernel(
    const Context& dev_ctx,
    const std::vector<const DenseTensor*>& params,
    const std::vector<const DenseTensor*>& grads,
    const std::vector<const DenseTensor*>& moments1,
    const std::vector<const DenseTensor*>& moments2,
    const paddle::optional<std::vector<const DenseTensor*>>& master_param,
    const DenseTensor& beta1_pow,
    const DenseTensor& beta2_pow,
    const DenseTensor& learning_rate,
    const paddle::optional<DenseTensor>& skip_update,
    const Scalar& beta1,
    const Scalar& beta2,
    const Scalar& epsilon,
    int chunk_size,
    float weight_decay,
    bool mode,
    bool multi_precision,
    bool use_global_beta_pow,
    std::vector<DenseTensor*> params_out,
    std::vector<DenseTensor*> moments1_out,
    std::vector<DenseTensor*> moments2_out,
    std::vector<DenseTensor*> master_param_out,
    DenseTensor* beta1_pow_out,
    DenseTensor* beta2_pow_out) {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;

  VLOG(4) << "use_global_beta_pow:" << use_global_beta_pow;
  MPDType beta1_ = beta1.to<MPDType>();
  MPDType beta2_ = beta2.to<MPDType>();
  MPDType weight_decay_ = static_cast<MPDType>(weight_decay);
  MPDType epsilon_ = epsilon.to<MPDType>();

  bool skip_update_ = false;
  if (skip_update.is_initialized()) {
    PADDLE_ENFORCE_EQ(
        skip_update->numel(),
        1,
        errors::InvalidArgument("Input(SkipUpdate) size must be 1, but get %d",
                                skip_update->numel()));
    std::vector<bool> skip_update_vec;
    paddle::framework::TensorToVector(*skip_update, dev_ctx, &skip_update_vec);
    skip_update_ = skip_update_vec[0];
  }

  // skip_update=true
  // mutable_data
  if (skip_update_) {
    VLOG(4) << "Adam skip update";
    for (int i = 0; i < params.size(); i++) {
      phi::Copy(dev_ctx, *params[i], dev_ctx.GetPlace(), false, params_out[i]);
      phi::Copy(
          dev_ctx, *moments1[i], dev_ctx.GetPlace(), false, moments1_out[i]);
      phi::Copy(
          dev_ctx, *moments2[i], dev_ctx.GetPlace(), false, moments2_out[i]);
    }
    phi::Copy(dev_ctx, beta1_pow, beta1_pow.place(), false, beta1_pow_out);
    phi::Copy(dev_ctx, beta2_pow, beta2_pow.place(), false, beta2_pow_out);
    return;
  }

  std::vector<std::vector<DenseTensor*>> tensor_lists;

  tensor_lists.push_back(params_out);
  tensor_lists.push_back(moments1_out);
  tensor_lists.push_back(moments2_out);
  if (multi_precision) {
    tensor_lists.push_back(master_param_out);
  }

  if (multi_precision) {
    multi_tensor_apply<5, MPDType>(dev_ctx,
                                   BLOCK_SIZE,
                                   chunk_size,
                                   tensor_lists,
                                   grads,
                                   AdamFunctor<T, MPDType, 5>(),
                                   beta1_,
                                   beta2_,
                                   beta1_pow.data<MPDType>(),
                                   beta2_pow.data<MPDType>(),
                                   epsilon_,
                                   learning_rate.data<MPDType>(),
                                   mode ? (adamMode_t)1 : (adamMode_t)0,
                                   multi_precision,
                                   weight_decay_);
  } else {
    multi_tensor_apply<4, MPDType>(dev_ctx,
                                   BLOCK_SIZE,
                                   chunk_size,
                                   tensor_lists,
                                   grads,
                                   AdamFunctor<T, MPDType, 4>(),
                                   beta1_,
                                   beta2_,
                                   beta1_pow.data<MPDType>(),
                                   beta2_pow.data<MPDType>(),
                                   epsilon_,
                                   learning_rate.data<MPDType>(),
                                   mode ? (adamMode_t)1 : (adamMode_t)0,
                                   multi_precision,
                                   weight_decay_);
  }

  if (!use_global_beta_pow) {
    // Update with gpu
    UpdateBetaPow<MPDType><<<1, 32, 0, dev_ctx.stream()>>>(
        beta1_,
        beta2_,
        beta1_pow.data<MPDType>(),
        beta2_pow.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(beta1_pow_out),
        dev_ctx.template Alloc<MPDType>(beta2_pow_out));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(multi_tensor_adam,
                   GPU,
                   ALL_LAYOUT,
                   phi::MultiTensorAdamKernel,
                   phi::dtype::float16,
                   float,
                   double) {}
