#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/fill_diagonal_kernel.h"

#include <algorithm>
#include <vector>

#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/common_shape.h"

namespace phi {

template <typename T>
__global__ void fill_constant_kernel(const int64_t featuresize,
                                     T* in_data,
                                     int64_t strides,
                                     int offset,
                                     T fillvar,
                                     int dims) {
  for (int64_t idx = blockIdx.x * featuresize + threadIdx.x;
       idx * strides + offset < (blockIdx.x + 1) * featuresize;
       idx += blockDim.x) {
    // to check if the new position with offset is still in the same line;
    // this modify should not affect across lines.
    // out_dims[1] is also work for tensor with dim>2, for which the dims must
    // be the same number
    if ((idx * strides) % dims + offset < dims &&
        (idx * strides) % dims + offset >= 0) {
      in_data[idx * strides + offset] = fillvar;
    }
  }
}

template <typename T, typename Context>
void FillDiagonalKernel(const Context& ctx,
                        const DenseTensor& x,
                        float value,
                        int offset,
                        bool wrap,
                        DenseTensor* out) {
#ifdef __HIPCC__
  const int64_t kMaxBlockDim = 256;
#else
  const int64_t kMaxBlockDim = 512;
#endif
  phi::Copy(ctx, x, ctx.GetPlace(), false, out);

  T* out_data = ctx.template Alloc<T>(out);
  auto fill_val = static_cast<T>(value);
  T temp_var = static_cast<T>(fill_val);

  auto size = out->numel();
  auto out_dims = out->dims();
  auto strides = funcs::CalStride(out_dims);

  // The wrap mode supported only the dims equels to 2; In wrap mode, the
  // value will be filled in cycles
  if (!wrap) {
    size = std::min(size, out_dims[1] * out_dims[1]);
  }

  int64_t kBlockDim = std::min(int64_t(size / strides), kMaxBlockDim);
  fill_constant_kernel<T><<<1, kBlockDim, 0>>>(
      size, out_data, strides, offset, temp_var, out_dims[1]);
}

}  // namespace phi

PD_REGISTER_KERNEL(fill_diagonal,
                   GPU,
                   ALL_LAYOUT,
                   phi::FillDiagonalKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   phi::dtype::float16,
                   bool) {}
