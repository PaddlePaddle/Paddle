#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/amp_kernel.h"

#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/impl/amp_kernel_impl.h"

#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/memory/memory.h"

namespace phi {

// Utils

template <typename T>
__global__ void InverseAndMemset(const T* s, T* o, bool* found_inf) {
  *o = 1.0 / *s;
  *found_inf = false;
}

template <typename T, typename MT>
__global__ void CheckFiniteAndUnscale(const T** xs,
                                      const MT* scale,
                                      int64_t size,
                                      int64_t* starts,
                                      bool* found_inf,
                                      T** outs) {
  const int64_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  // copy starts array from global memory to shared memory
  extern __shared__ int64_t s_starts[];
  for (int i = threadIdx.x; i <= size; i += blockDim.x) {
    s_starts[i] = starts[i];
  }
  __syncthreads();

  const int64_t num = s_starts[size];
  int xs_index = 0;
  bool local_found_inf = false;
  const MT local_scale = *scale;
  for (int64_t idx = tid; idx < num; idx += gridDim.x * blockDim.x) {
    // get the "out" index of "id"
    // For example:
    // idx = 15, starts = [0, 10, 10, 20, 30]
    // because 10 <= idx < 20 ==>
    // the idx element locate in the 3rd tensor (notice the 2nd tensor size is
    // 0)
    int next_xs_index = xs_index;
    while (idx >= s_starts[next_xs_index]) next_xs_index++;
    xs_index = next_xs_index - 1;

    // get in data and out data
    const T* in = xs[xs_index];
    T* out = outs[xs_index];
    int64_t in_idx = idx - s_starts[xs_index];

    // Unscale
    MT val = static_cast<MT>(in[in_idx]) * local_scale;
    T narrow_val = static_cast<T>(val);
    out[in_idx] = narrow_val;

    // CheckFinite
    if (!isfinite(narrow_val)) {
      local_found_inf = true;
    }
  }
  if (local_found_inf) {
    *found_inf = true;
  }
}

template <typename T, typename FoundNanInfFlagT>
__global__ void GpuUpdateLossScaling(const FoundNanInfFlagT found_inf_data,
                                     const T* pre_loss_scaling_data,
                                     const int* good_in_data,
                                     const int* bad_in_data,
                                     const int incr_every_n_steps,
                                     const int decr_every_n_nan_or_inf,
                                     const float incr_ratio,
                                     const float decr_ratio,
                                     T* updated_loss_scaling_data,
                                     int* good_out_data,
                                     int* bad_out_data) {
  Update<T>(found_inf_data,
            pre_loss_scaling_data,
            good_in_data,
            bad_in_data,
            incr_every_n_steps,
            decr_every_n_nan_or_inf,
            incr_ratio,
            decr_ratio,
            updated_loss_scaling_data,
            good_out_data,
            bad_out_data);
}

template <typename T>
__global__ void FusedFillIf(T** outs,
                            const size_t xs_size,
                            const int64_t* starts,
                            const T value,
                            const bool* has_inf) {
  if (!(*has_inf)) return;

  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // copy starts array from global memory to shared memory
  extern __shared__ int64_t s_starts[];
  for (int i = threadIdx.x; i <= xs_size; i += blockDim.x) {
    s_starts[i] = starts[i];
  }
  __syncthreads();

  const int64_t total_num = s_starts[xs_size];
  int out_index = 0;

  for (int64_t id = tid; id < total_num; id += blockDim.x * gridDim.x) {
    // get the "out" index of "id"
    // For example:
    // id = 15, starts = [0, 10, 10, 20, 30]
    // because 10 <= id < 20 ==>
    // the id element locate in the 3rd tensor (notice the 2nd tensor size is 0)
    int next_out_index = out_index;
    while (id >= s_starts[next_out_index]) next_out_index++;
    out_index = next_out_index - 1;

    // get data pointer and index
    T* out_data = outs[out_index];
    int64_t idx = id - s_starts[out_index];

    // set value
    out_data[idx] = value;
  }
}

template <typename T>
class LazyZeros<phi::GPUContext, T> {
 public:
  void operator()(const phi::GPUContext& dev_ctx,
                  const bool* found_inf_data,
                  const std::vector<const DenseTensor*>& xs,
                  const std::vector<DenseTensor*>& outs) {
    size_t xs_size = xs.size();
    if (xs_size == 0) return;

    const auto& cpu_place = phi::CPUPlace();
    // alloc each tensor's start index and copy to device
    auto h_in_starts_mem =
        paddle::memory::Alloc(cpu_place, (xs_size + 1) * sizeof(int64_t));
    int64_t* h_starts = reinterpret_cast<int64_t*>(h_in_starts_mem->ptr());

    auto d_in_starts_mem = paddle::memory::Alloc(
        dev_ctx.GetPlace(),
        (xs_size + 1) * sizeof(int64_t),
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
    int64_t* d_starts = reinterpret_cast<int64_t*>(d_in_starts_mem->ptr());

    // the start index value of each tensor is
    // the sum of previous tensor's size. For example:
    // outs = [10, 0, 10, 10] ==> starts = [0, 10, 10, 20, 30]
    h_starts[0] = 0;
    for (int i = 0; i < xs_size; i++) {
      h_starts[i + 1] = h_starts[i] + outs[i]->numel();
    }
    paddle::memory::Copy(dev_ctx.GetPlace(),
                         d_starts,
                         cpu_place,
                         h_starts,
                         (xs_size + 1) * sizeof(int64_t),
                         dev_ctx.stream());

    // copy each tensor of "outs" data address array to device
    auto h_out_addrs_mem =
        paddle::memory::Alloc(cpu_place, xs_size * sizeof(T*));
    T** h_out_addrs = reinterpret_cast<T**>(h_out_addrs_mem->ptr());

    auto d_out_addrs_mem = paddle::memory::Alloc(
        dev_ctx.GetPlace(),
        xs_size * sizeof(T*),
        phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
    T** d_out_addrs = reinterpret_cast<T**>(d_out_addrs_mem->ptr());

    for (size_t i = 0; i < xs_size; ++i) {
      h_out_addrs[i] = dev_ctx.Alloc<T>(outs[i]);
    }
    paddle::memory::Copy(dev_ctx.GetPlace(),
                         d_out_addrs,
                         cpu_place,
                         h_out_addrs,
                         xs_size * sizeof(T*),
                         dev_ctx.stream());

    // launch cuda kernel
    int64_t total_num = h_starts[xs_size];
    int64_t threads_per_block = std::min(static_cast<int64_t>(1024), total_num);
    int64_t elements_per_block =
        threads_per_block * 50;  // each thread deal with 50 data
    int64_t blocks_per_grid =
        (total_num + elements_per_block - 1) / elements_per_block;
    FusedFillIf<T><<<blocks_per_grid,
                     threads_per_block,
                     (xs_size + 1) * sizeof(int64_t),
                     dev_ctx.stream()>>>(
        d_out_addrs, xs_size, d_starts, static_cast<T>(0), found_inf_data);
  }
};

template <typename T, bool IsFoundInfOnCPU>
class UpdateLossScalingFunctor<phi::GPUContext, T, IsFoundInfOnCPU> {
 public:
  void operator()(const phi::GPUContext& dev_ctx,
                  const bool* found_inf_data,
                  const T* pre_loss_scaling_data,
                  const int* good_in_data,
                  const int* bad_in_data,
                  const int incr_every_n_steps,
                  const int decr_every_n_nan_or_inf,
                  const float incr_ratio,
                  const float decr_ratio,
                  T* updated_loss_scaling_data,
                  int* good_out_data,
                  int* bad_out_data) const {
    if (IsFoundInfOnCPU) {
      GpuUpdateLossScaling<T>
          <<<1, 1, 0, dev_ctx.stream()>>>(*found_inf_data,
                                          pre_loss_scaling_data,
                                          good_in_data,
                                          bad_in_data,
                                          incr_every_n_steps,
                                          decr_every_n_nan_or_inf,
                                          incr_ratio,
                                          decr_ratio,
                                          updated_loss_scaling_data,
                                          good_out_data,
                                          bad_out_data);
    } else {
      GpuUpdateLossScaling<T>
          <<<1, 1, 0, dev_ctx.stream()>>>(found_inf_data,
                                          pre_loss_scaling_data,
                                          good_in_data,
                                          bad_in_data,
                                          incr_every_n_steps,
                                          decr_every_n_nan_or_inf,
                                          incr_ratio,
                                          decr_ratio,
                                          updated_loss_scaling_data,
                                          good_out_data,
                                          bad_out_data);
    }
  }
};

// Kernels

template <typename T, typename Context>
void CheckFiniteAndUnscaleKernel(const Context& dev_ctx,
                                 const std::vector<const DenseTensor*>& xs,
                                 const DenseTensor& scale,
                                 std::vector<DenseTensor*> outs,
                                 DenseTensor* found_infinite) {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;

  const MPDType* scale_data = scale.data<MPDType>();
  bool* found_inf_data = dev_ctx.template Alloc<bool>(found_infinite);

  DenseTensor inverse_scale = Empty<MPDType>(dev_ctx, {1});
  MPDType* inverse_scale_v = inverse_scale.template data<MPDType>();

  InverseAndMemset<MPDType><<<1, 1, 0, dev_ctx.stream()>>>(
      scale_data, inverse_scale_v, found_inf_data);

  size_t xs_size = xs.size();
  if (xs_size == 0) return;

  const auto& cpu_place = phi::CPUPlace();
  // calculate each tensor's start index and copy to device
  auto h_starts_tensor =
      paddle::memory::Alloc(cpu_place, (xs_size + 1) * sizeof(int64_t));
  int64_t* h_starts = reinterpret_cast<int64_t*>(h_starts_tensor->ptr());

  auto d_starts_tensor = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      (xs_size + 1) * sizeof(int64_t),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  int64_t* d_starts = reinterpret_cast<int64_t*>(d_starts_tensor->ptr());

  // the start index value of each tensor is
  // the sum of previous tensor's size. For example:
  // x = [10, 0, 10, 10] ==> starts = [0, 10, 10, 20, 30]
  h_starts[0] = 0;
  for (int i = 1; i <= xs_size; i++) {
    h_starts[i] = h_starts[i - 1] + xs[i - 1]->numel();
  }
  int64_t total_num = h_starts[xs_size];
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       d_starts,
                       cpu_place,
                       h_starts,
                       (xs_size + 1) * sizeof(int64_t),
                       dev_ctx.stream());

  // copy each tensor's data address to device
  auto h_mem = paddle::memory::Alloc(cpu_place, 2 * xs_size * sizeof(T*));
  const T** h_xs = reinterpret_cast<const T**>(h_mem->ptr());
  T** h_outs = reinterpret_cast<T**>(h_mem->ptr()) + xs_size;

  auto d_mem = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      2 * xs_size * sizeof(T*),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  const T** d_xs = reinterpret_cast<const T**>(d_mem->ptr());
  T** d_outs = reinterpret_cast<T**>(d_mem->ptr()) + xs_size;

  for (size_t i = 0; i < xs_size; ++i) {
    h_xs[i] = xs[i]->data<T>();
    h_outs[i] = dev_ctx.template Alloc<T>(outs[i]);
  }
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       d_xs,
                       cpu_place,
                       h_xs,
                       2 * xs_size * sizeof(T*),
                       dev_ctx.stream());

  // Launch Kernel
  int threads_per_block = std::min(static_cast<int64_t>(1024), total_num);
  int elements_per_block =
      threads_per_block * 20;  // each thread deal with 20 number
  int blocks_per_grid =
      (total_num + elements_per_block - 1) / elements_per_block;
  CheckFiniteAndUnscale<T, MPDType><<<blocks_per_grid,
                                      threads_per_block,
                                      (xs_size + 1) * sizeof(int64_t),
                                      dev_ctx.stream()>>>(
      d_xs, inverse_scale_v, xs_size, d_starts, found_inf_data, d_outs);
}

}  // namespace phi

PD_REGISTER_KERNEL(check_finite_and_unscale,
                   GPU,
                   ALL_LAYOUT,
                   phi::CheckFiniteAndUnscaleKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(update_loss_scaling,
                   GPU,
                   ALL_LAYOUT,
                   phi::UpdateLossScalingKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  kernel->InputAt(1).SetBackend(phi::Backend::ALL_BACKEND);
}
