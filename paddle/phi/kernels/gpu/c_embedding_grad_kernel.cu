#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/c_embedding_kernel.h"

#include "glog/logging.h"

#include "paddle/common/flags.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/embedding_grad.h"

PD_DECLARE_int64(embedding_deterministic);

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename IndexT>
__global__ void CEmbeddingGrad(T* table,
                               const T* output,
                               const IndexT* ids,
                               const int rows,
                               const int columns,
                               const int64_t N,
                               const int64_t start_idx,
                               const int64_t end_idx,
                               const int64_t limit) {
  CUDA_KERNEL_LOOP(i, limit) {
    size_t row = i / columns;
    size_t col = i % columns;
    auto id = ids[row];
    if (id >= start_idx && id < end_idx) {
      auto real_idx = id - start_idx;
      phi::CudaAtomicAdd(&table[real_idx * columns + col], output[i]);
    }
  }
}

template <typename T, typename Context>
void CEmbeddingGradKernel(const Context& dev_ctx,
                          const DenseTensor& w,
                          const DenseTensor& ids,
                          const DenseTensor& out_grad,
                          int64_t start_index,
                          DenseTensor* w_grad) {
  int N = w_grad->dims()[0];
  int D = w_grad->dims()[1];
  int K = ids.numel();

  auto limit = K * D;
  int blocks = NumBlocks(limit);
  int threads = kNumCUDAThreads;

  const T* d_output = out_grad.data<T>();
  T* d_table = dev_ctx.template Alloc<T>(w_grad);

  auto t = EigenVector<T>::Flatten(*w_grad);
  t.device(*dev_ctx.eigen_device()) = t.constant(static_cast<T>(0));

  const auto& index_type = ids.dtype();
  if (FLAGS_embedding_deterministic == 1) {
    if (index_type == phi::DataType::INT32) {
      phi::funcs::LaunchEmbeddingGradDeterministicKernel<T, int32_t>(
          dev_ctx,
          ids.data<int32_t>(),
          d_output,
          d_table,
          N,
          D,
          K,
          start_index);
      return;
    } else if (index_type == phi::DataType::INT64) {
      phi::funcs::LaunchEmbeddingGradDeterministicKernel<T, int64_t>(
          dev_ctx,
          ids.data<int64_t>(),
          d_output,
          d_table,
          N,
          D,
          K,
          start_index);
      return;
    }
  } else {
    if (FLAGS_embedding_deterministic > 1) {
      VLOG(2) << "Run grad kernel of embedding with single thread.";
      blocks = 1;
    }
    const int64_t end_idx = start_index + N;
    if (index_type == phi::DataType::INT32) {
      CEmbeddingGrad<T, int32_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(d_table,
                                                     d_output,
                                                     ids.data<int32_t>(),
                                                     K,
                                                     D,
                                                     N,
                                                     start_index,
                                                     end_idx,
                                                     limit);
      return;
    } else if (index_type == phi::DataType::INT64) {
      CEmbeddingGrad<T, int64_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(d_table,
                                                     d_output,
                                                     ids.data<int64_t>(),
                                                     K,
                                                     D,
                                                     N,
                                                     start_index,
                                                     end_idx,
                                                     limit);
      return;
    }
  }
  PADDLE_THROW(phi::errors::InvalidArgument(
      "The data type of Input(Ids) must be int32 or int64."));
}

}  // namespace phi

#if NCCL_VERSION_CODE >= 21000 && CUDA_VERSION >= 11000
PD_REGISTER_KERNEL(c_embedding_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CEmbeddingGradKernel,
                   float,
                   double,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(c_embedding_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CEmbeddingGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#endif
