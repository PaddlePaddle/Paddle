// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/squeeze_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/squeeze_kernel_impl.h"

PD_REGISTER_KERNEL(squeeze,
                   GPU,
                   ALL_LAYOUT,
                   phi::SqueezeKernel,
                   float,
                   double,
                   phi::dtype::bfloat16,
                   phi::dtype::float16,
                   bool,
                   int,
                   uint8_t,
                   int8_t,
                   int64_t,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(squeeze_with_xshape,
                   GPU,
                   ALL_LAYOUT,
                   phi::SqueezeWithXShapeKernel,
                   float,
                   double,
                   phi::dtype::bfloat16,
                   phi::dtype::float16,
                   bool,
                   int,
                   uint8_t,
                   int8_t,
                   int64_t,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
