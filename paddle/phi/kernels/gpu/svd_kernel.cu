#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifndef PADDLE_WITH_HIP
// HIP not support cusolver

#include "paddle/phi/kernels/svd_kernel.h"

#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/backends/dynload/cusolver.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"
#include "paddle/phi/kernels/transpose_kernel.h"

namespace phi {

template <class T>
static void GesvdjBatched(const phi::GPUContext& dev_ctx,
                          int batchSize,
                          int m,
                          int n,
                          int k,
                          T* A,
                          T* U,
                          T* V,
                          T* S,
                          int* info,
                          int thin_UV = 1);

template <>
void GesvdjBatched<float>(const phi::GPUContext& dev_ctx,
                          int batchSize,
                          int m,
                          int n,
                          int k,
                          float* A,
                          float* U,
                          float* V,
                          float* S,
                          int* info,
                          int thin_UV) {
  /* compute singular vectors */
  const hipsolverEigMode_t jobz =
      HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */
  hipsolverGesvdjInfo_t gesvdj_params = NULL;
  int lda = m;
  int ldu = m;
  int ldt = n;
  int lwork = 0;
  auto handle = dev_ctx.cusolver_dn_handle();
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipsolverDnSgesvdj_bufferSize(handle,
                                                 jobz,
                                                 thin_UV,
                                                 m,
                                                 n,
                                                 A,
                                                 lda,
                                                 S,
                                                 U,
                                                 ldu,
                                                 V,
                                                 ldt,
                                                 &lwork,
                                                 gesvdj_params));
  auto workspace = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      lwork * sizeof(float),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  float* workspace_ptr = reinterpret_cast<float*>(workspace->ptr());
  int stride_A = lda * n;
  int stride_U = ldu * (thin_UV ? k : m);
  int stride_V = ldt * (thin_UV ? k : n);
  for (int i = 0; i < batchSize; ++i) {
    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipsolverDnSgesvdj(handle,
                                                               jobz,
                                                               thin_UV,
                                                               m,
                                                               n,
                                                               A + stride_A * i,
                                                               lda,
                                                               S + k * i,
                                                               U + stride_U * i,
                                                               ldu,
                                                               V + stride_V * i,
                                                               ldt,
                                                               workspace_ptr,
                                                               lwork,
                                                               info,
                                                               gesvdj_params));
    // check the error info
    int error_info;
    paddle::memory::Copy(phi::CPUPlace(),
                         &error_info,
                         dev_ctx.GetPlace(),
                         info,
                         sizeof(int),
                         dev_ctx.stream());
    PADDLE_ENFORCE_EQ(
        error_info,
        0,
        phi::errors::PreconditionNotMet(
            "For batch [%d]: CUSolver SVD is not zero. [%d]", i, error_info));
  }
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

template <>
void GesvdjBatched<double>(const phi::GPUContext& dev_ctx,
                           int batchSize,
                           int m,
                           int n,
                           int k,
                           double* A,
                           double* U,
                           double* V,
                           double* S,
                           int* info,
                           int thin_UV) {
  /* compute singular vectors */
  const hipsolverEigMode_t jobz =
      HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */
  hipsolverGesvdjInfo_t gesvdj_params = NULL;
  int lda = m;
  int ldu = m;
  int ldt = n;
  int lwork = 0;
  auto handle = dev_ctx.cusolver_dn_handle();
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipsolverDnDgesvdj_bufferSize(handle,
                                                 jobz,
                                                 thin_UV,
                                                 m,
                                                 n,
                                                 A,
                                                 lda,
                                                 S,
                                                 U,
                                                 ldu,
                                                 V,
                                                 ldt,
                                                 &lwork,
                                                 gesvdj_params));
  auto workspace = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      lwork * sizeof(double),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  double* workspace_ptr = reinterpret_cast<double*>(workspace->ptr());
  int stride_A = lda * n;
  int stride_U = ldu * (thin_UV ? k : m);
  int stride_V = ldt * (thin_UV ? k : n);
  for (int i = 0; i < batchSize; ++i) {
    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::hipsolverDnDgesvdj(handle,
                                                               jobz,
                                                               thin_UV,
                                                               m,
                                                               n,
                                                               A + stride_A * i,
                                                               lda,
                                                               S + k * i,
                                                               U + stride_U * i,
                                                               ldu,
                                                               V + stride_V * i,
                                                               ldt,
                                                               workspace_ptr,
                                                               lwork,
                                                               info,
                                                               gesvdj_params));
    // check the error info
    int error_info;
    paddle::memory::Copy(phi::CPUPlace(),
                         &error_info,
                         dev_ctx.GetPlace(),
                         info,
                         sizeof(int),
                         dev_ctx.stream());
    PADDLE_ENFORCE_EQ(
        error_info,
        0,
        phi::errors::PreconditionNotMet(
            "For batch [%d]: CUSolver SVD is not zero. [%d]", i, error_info));
  }
  PADDLE_ENFORCE_GPU_SUCCESS(
      phi::dynload::hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

template <typename T, typename Context>
void SvdKernel(const Context& dev_ctx,
               const DenseTensor& X,
               bool full_matrices,
               DenseTensor* U,
               DenseTensor* S,
               DenseTensor* VH) {
  auto& dims = X.dims();
  int batch_count = 1;
  for (int i = 0; i < dims.size() - 2; i++) {
    batch_count *= dims[i];
  }
  int rank = dims.size();
  int m = dims[rank - 2];
  int n = dims[rank - 1];

  auto* u_data = dev_ctx.template Alloc<phi::dtype::Real<T>>(U);
  auto* vh_data = dev_ctx.template Alloc<phi::dtype::Real<T>>(VH);
  auto* s_data = dev_ctx.template Alloc<phi::dtype::Real<T>>(S);
  // NOTE:(@xiongkun03)
  // matrices are assumed to be stored in column-major order in cusolver
  // then view A as n x m and do A^T SVD, we can avoid transpose
  // Must Copy X once, because the gesvdj will change the origin input matrix
  DenseTensor x_tmp;
  Copy(dev_ctx, X, dev_ctx.GetPlace(), false, &x_tmp);
  auto info = Empty<int, Context>(dev_ctx, {batch_count});
  int* info_ptr = reinterpret_cast<int*>(info.data());

  GesvdjBatched<T>(dev_ctx,
                   batch_count,
                   n,
                   m,
                   std::min(m, n),
                   dev_ctx.template Alloc<T>(&x_tmp),
                   vh_data,
                   u_data,
                   s_data,
                   info_ptr,
                   !full_matrices);

  auto UT_dim = U->dims();
  std::swap(UT_dim[rank - 1], UT_dim[rank - 2]);  // Get the dim of UT_dim
  U->Resize(UT_dim);                              // U is entirely UT
  auto tmp_U = TransposeLast2Dim<T>(dev_ctx, *U);
  U->ShareDataWith(tmp_U);  // U becomse UT, aka VT;
}
}  // namespace phi

PD_REGISTER_KERNEL(svd,  // cuda_only
                   GPU,
                   ALL_LAYOUT,
                   phi::SvdKernel,
                   float,
                   double) {}

#endif  // not PADDLE_WITH_HIP
