#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flip_kernel.h"

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/place.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T>
__global__ void flip_cuda_kernel(const int N,
                                 const T* in_data,
                                 T* out_data,
                                 int64_t* x_shape,
                                 int64_t* x_stride,
                                 int* flip_dims,
                                 int flip_dims_size,
                                 int total_dims) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int cur_indices = idx, rem = 0, dst_offset = 0;
  for (int i = 0; i < total_dims; ++i) {
    int64_t temp = cur_indices;
    cur_indices = cur_indices / x_stride[i];
    rem = temp - cur_indices * x_stride[i];
    // flip the indices if it is in flip_dims
    for (int j = 0; j < flip_dims_size; ++j) {
      if (i == flip_dims[j]) {
        cur_indices = x_shape[i] - 1 - cur_indices;
      }
    }
    dst_offset += cur_indices * x_stride[i];
    cur_indices = rem;
  }
  out_data[idx] = in_data[dst_offset];
}

template <typename T, typename Context>
void FlipKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const std::vector<int>& axis,
                DenseTensor* out) {
  const auto gplace = dev_ctx.GetPlace();
  auto cplace = phi::CPUPlace();
  std::vector<int> flip_dims = axis;

  auto* in_data = x.data<T>();
  auto* out_data = dev_ctx.template Alloc<T>(out);

  const int flip_dims_size = static_cast<int>(flip_dims.size());
  auto x_dims = x.dims();
  const int total_dims = x_dims.size();
  const int N = x.numel();

  int block_size = 512;
  dim3 dim_block(block_size);
  dim3 dim_grid((N + block_size - 1) / block_size);

  for (size_t i = 0; i < flip_dims.size(); ++i) {
    if (flip_dims[i] < 0) {
      flip_dims[i] += total_dims;
    }
  }

  auto x_stride = phi::stride(x_dims);
  std::vector<int64_t> x_dims_v = phi::vectorize(x_dims);
  std::vector<int64_t> x_stride_v = phi::vectorize(x_stride);

  int bytes = total_dims * sizeof(int64_t);
  auto x_strides_array_tmp = paddle::memory::Alloc(dev_ctx, bytes);
  int64_t* x_strides_array_gpu =
      reinterpret_cast<int64_t*>(x_strides_array_tmp->ptr());
  paddle::memory::Copy(gplace,
                       x_strides_array_gpu,
                       cplace,
                       x_stride_v.data(),
                       bytes,
                       dev_ctx.stream());

  auto x_shape_array_tmp = paddle::memory::Alloc(dev_ctx, bytes);
  int64_t* x_shape_array_gpu =
      reinterpret_cast<int64_t*>(x_shape_array_tmp->ptr());
  paddle::memory::Copy(gplace,
                       x_shape_array_gpu,
                       cplace,
                       x_dims_v.data(),
                       bytes,
                       dev_ctx.stream());

  bytes = flip_dims_size * sizeof(int);
  auto flip_dims_array_tmp = paddle::memory::Alloc(dev_ctx, bytes);
  int* flip_dims_array_gpu = reinterpret_cast<int*>(flip_dims_array_tmp->ptr());
  paddle::memory::Copy(gplace,
                       flip_dims_array_gpu,
                       cplace,
                       flip_dims.data(),
                       bytes,
                       dev_ctx.stream());

  flip_cuda_kernel<T><<<dim_grid, dim_block, 0, dev_ctx.stream()>>>(
      N,
      in_data,
      out_data,
      x_shape_array_gpu,
      x_strides_array_gpu,
      flip_dims_array_gpu,
      flip_dims_size,
      total_dims);
}
}  // namespace phi

PD_REGISTER_KERNEL(flip,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlipKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int,
                   int64_t,
                   bool,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
