#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include "paddle/phi/core/device_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gather_tree_kernel.h"

namespace phi {

template <typename T>
__global__ void GatherTree(const T *ids_data,
                           const T *parents_data,
                           T *out_data,
                           const int64_t max_length,
                           const int64_t batch_size,
                           const int64_t beam_size) {
  CUDA_KERNEL_LOOP(i, batch_size * beam_size) {
    int batch = i / beam_size;
    int beam = i % beam_size;
    auto idx =
        (max_length - 1) * batch_size * beam_size + batch * beam_size + beam;
    out_data[idx] = ids_data[idx];
    auto parent = parents_data[idx];
    for (int step = max_length - 2; step >= 0; step--) {
      idx = step * batch_size * beam_size + batch * beam_size;
      out_data[idx + beam] = ids_data[idx + parent];
      parent = parents_data[idx + parent];
    }
  }
}

template <typename T, typename Context>
void GatherTreeKernel(const Context &dev_ctx,
                      const DenseTensor &ids,
                      const DenseTensor &parents,
                      DenseTensor *out) {
  const auto *ids_data = ids.data<T>();
  const auto *parents_data = parents.data<T>();
  T *out_data = dev_ctx.template Alloc<T>(out);

  PADDLE_ENFORCE_NOT_NULL(ids_data,
                          phi::errors::InvalidArgument(
                              "Input(Ids) of gather_tree should not be null."));

  PADDLE_ENFORCE_NOT_NULL(
      parents_data,
      phi::errors::InvalidArgument(
          "Input(Parents) of gather_tree should not be null."));

  auto &ids_dims = ids.dims();
  int64_t max_length = ids_dims[0];
  int64_t batch_size = ids_dims[1];
  int64_t beam_size = ids_dims[2];

  const int block = 512;
  int max_threads =
      std::min(static_cast<int64_t>(dev_ctx.GetMaxPhysicalThreadCount()),
               batch_size * beam_size);
  const int grid = std::max(max_threads / block, 1);
  GatherTree<<<grid, block>>>(
      ids_data, parents_data, out_data, max_length, batch_size, beam_size);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    gather_tree, GPU, ALL_LAYOUT, phi::GatherTreeKernel, int, int64_t) {}
