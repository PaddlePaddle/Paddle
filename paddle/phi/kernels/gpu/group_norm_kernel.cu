#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/group_norm_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/group_norm_utils.h"

namespace phi {

template <typename T>
__global__ void GroupNormForwardGetMeanAndVar(const T* x,
                                              int N,
                                              int C,
                                              int W,
                                              int imsize,
                                              int groups,
                                              int group_size,
                                              T* mean,
                                              T* var) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  T x_mean = 0, x_var = 0;
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    int hid = imid / W;
    int wid = imid % W;
    val = x[(bid * H + hid) * W * C + wid * C + ccid];

    x_mean += val;
    x_var += val * val;
  }
  x_mean /= number * imsize;
  x_var /= number * imsize;
  CudaAtomicAddWithWarp(&mean[bid * groups + gid], x_mean);
  CudaAtomicAddWithWarp(&var[bid * groups + gid], x_var);
}

template <typename T, int flags>
__global__ void GroupNormForward(const T* x,
                                 const T* mean,
                                 const T* var,
                                 const T* scale,
                                 const T* bias,
                                 int N,
                                 int C,
                                 int W,
                                 int imsize,
                                 int groups,
                                 int group_size,
                                 T epsilon,
                                 T* y,
                                 T* real_var,
                                 const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  auto ng = bid * groups + gid;
  T x_mean = mean[ng];
  T x_var = var[ng];
  x_var = x_var - x_mean * x_mean;
  T var_inv = rsqrt(x_var + epsilon);
  if (cid == 0 && threadIdx.x == 0) {
    real_var[ng] = x_var;
  }
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    int hid, wid;
    int index = (bid * C + ccid) * imsize + imid;
    if (data_layout == DataLayout::kNCHW) {
      val = x[index];
    } else {
      hid = imid / W;
      wid = imid % W;
      val = x[(bid * H + hid) * W * C + wid * C + ccid];
    }
    val = (val - x_mean) * var_inv;
    if (flags & kHasScale) {
      val *= scale[ccid];
    }
    if (flags & kHasBias) {
      val += bias[ccid];
    }
    if (data_layout == DataLayout::kNCHW) {
      y[index] = val;
    } else {
      y[(bid * H + hid) * W * C + wid * C + ccid] = val;
    }
  }
}

template <typename T, typename Context>
void GroupNormKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const paddle::optional<DenseTensor>& scale,
                     const paddle::optional<DenseTensor>& bias,
                     float epsilon,
                     int groups,
                     const std::string& data_layout_str,
                     DenseTensor* y,
                     DenseTensor* mean,
                     DenseTensor* var) {
  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_layout_str);
  const auto scale_ptr = scale.get_ptr();
  const auto bias_ptr = bias.get_ptr();

  const auto x_dims = x.dims();
  const int C = (data_layout == DataLayout::kNCHW ? x_dims[1]
                                                  : x_dims[x_dims.size() - 1]);
  const int group_size = C / groups;

  const int W = (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                                  : x_dims[x_dims.size() - 2]);

  dev_ctx.template Alloc<T>(y);
  dev_ctx.template Alloc<T>(mean);
  dev_ctx.template Alloc<T>(var);
  phi::funcs::SetConstant<GPUContext, T> set_zero;
  DenseTensor temp_var;
  temp_var.Resize(var->dims());
  dev_ctx.template Alloc<T>(&temp_var);
  auto* x_data = x.data<T>();
  auto* y_data = y->data<T>();
  auto* mean_data = mean->data<T>();
  auto* var_data = var->data<T>();
  auto* temp_var_data = temp_var.data<T>();

  const T* scale_data = nullptr;
  if (scale_ptr) scale_data = scale_ptr->data<T>();
  const T* bias_data = nullptr;
  if (bias_ptr) bias_data = bias_ptr->data<T>();

  int imsize = 1;
  if (data_layout == DataLayout::kNCHW) {
    for (int i = 2; i < x_dims.size(); ++i) {
      imsize *= x_dims[i];
    }
  } else {
    for (int i = 1; i < x_dims.size() - 1; ++i) {
      imsize *= x_dims[i];
    }
  }

#ifdef __HIPCC__
  int block_size = std::max(std::min(256, imsize), 64);
#else
  int block_size = std::min(1024, imsize);
#endif

  dim3 grid(group_size, groups, x_dims[0]);
  dim3 threads(block_size, 1, 1);
  if (data_layout == DataLayout::kNCHW) {
    using AccT = typename kps::details::MPTypeTrait<T>::Type;
    constexpr int vec_size = sizeof(float4) / sizeof(T);
    int size = group_size * imsize;
    const int max_num_threads = 1024;
    int max_block_size = std::min(size / vec_size, max_num_threads);
    int block_size_nchw = 1;
    while (block_size_nchw < max_block_size) {
      block_size_nchw *= 2;
    }
    block_size_nchw = std::max(block_size_nchw, kps::details::kWarpSize);
    dim3 grids(x_dims[0] * groups);
    dim3 blocks(block_size_nchw);
    if (size < vec_size * block_size_nchw) {
      ScalarGetMeanAndVarNCHW<T><<<grids, blocks, 0, dev_ctx.stream()>>>(
          x_data, mean_data, temp_var_data, size);
    } else {
      VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
          <<<grids, blocks, 0, dev_ctx.stream()>>>(
              x_data, mean_data, temp_var_data, size);
    }
  } else {
    set_zero(dev_ctx, mean, static_cast<T>(0));
    set_zero(dev_ctx, &temp_var, static_cast<T>(0));
    GroupNormForwardGetMeanAndVar<T>
        <<<grid, threads, 0, dev_ctx.stream()>>>(x_data,
                                                 x_dims[0],
                                                 C,
                                                 W,
                                                 imsize,
                                                 groups,
                                                 group_size,
                                                 mean_data,
                                                 temp_var_data);
  }
  int flags =
      (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
  UNROLL_ALL_CASES(flags,
                   GroupNormForward,
                   x_data,
                   mean_data,
                   temp_var_data,
                   scale_data,
                   bias_data,
                   x_dims[0],
                   C,
                   W,
                   imsize,
                   groups,
                   group_size,
                   epsilon,
                   y_data,
                   var_data,
                   data_layout);
}

template <typename T>
void GroupNormDirectCUDAFunctor<T>::operator()(gpuStream_t stream,
                                               const T* input,
                                               std::vector<int> input_shape,
                                               const T* bias,
                                               const T* scale,
                                               T* temp_mean,
                                               T* temp_variance,
                                               int groups,
                                               float eps,
                                               T* output,
                                               T* mean,
                                               T* variance,
                                               const DataLayout data_layout) {
  const auto input_ddim = phi::make_ddim(input_shape);
  const int C =
      (data_layout == DataLayout::kNCHW ? input_ddim[1]
                                        : input_ddim[input_ddim.size() - 1]);
  const int group_size = C / groups;
  const int W =
      (data_layout == DataLayout::kNCHW ? input_ddim[input_ddim.size() - 1]
                                        : input_ddim[input_ddim.size() - 2]);

  int image_size = 1;
  if (data_layout == DataLayout::kNCHW) {
    for (int i = 2; i < input_ddim.size(); ++i) {
      image_size *= input_ddim[i];
    }
  } else {
    for (int i = 1; i < input_ddim.size() - 1; ++i) {
      image_size *= input_ddim[i];
    }
  }
#ifdef __HIPCC__
  int block_size = std::max(std::min(256, image_size), 64);
#else
  int block_size = std::min(1024, image_size);
#endif
  dim3 grid(group_size, groups, input_ddim[0]);
  dim3 threads(block_size, 1, 1);
  if (data_layout == DataLayout::kNCHW) {
    using AccT = typename phi::kps::details::MPTypeTrait<float>::Type;
    constexpr int vec_size = sizeof(float4) / sizeof(float);
    int size = group_size * image_size;  // group element size
    const int max_num_threads = 1024;
    int max_block_size = std::min(size / vec_size, max_num_threads);
    int block_size_nchw = 1;
    while (block_size_nchw < max_block_size) {
      block_size_nchw *= 2;
    }

    block_size_nchw = std::max(block_size_nchw, phi::kps::details::kWarpSize);
    dim3 grids(input_ddim[0] * groups);
    dim3 blocks(block_size_nchw);

    if (size < vec_size * block_size_nchw) {
      phi::ScalarGetMeanAndVarNCHW<T>
          <<<grids, blocks, 0, stream>>>(input, temp_mean, temp_variance, size);
    } else {
      phi::VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
          <<<grids, blocks, 0, stream>>>(input, temp_mean, temp_variance, size);
    }
  } else {
    phi::GroupNormForwardGetMeanAndVar<T>
        <<<grid, threads, 0, stream>>>(input,
                                       input_ddim[0],
                                       C,
                                       W,
                                       image_size,
                                       groups,
                                       group_size,
                                       temp_mean,
                                       temp_variance);
  }
  GroupNormForward<T, 3><<<grid, threads, 0, stream>>>(
      input,
      temp_mean,
      temp_variance,
      scale,
      bias,
      input_ddim[0],
      C,
      W,
      image_size,
      groups,
      group_size,
      eps,
      output,
      variance,
      data_layout);  // for now, we only support nchw for group norm
}
template class GroupNormDirectCUDAFunctor<float>;
}  // namespace phi

PD_REGISTER_KERNEL(
    group_norm, GPU, ALL_LAYOUT, phi::GroupNormKernel, float, double) {}
