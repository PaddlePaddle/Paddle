#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/group_norm_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/layout.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/group_norm_utils.h"

namespace phi {
#define CHECK_CASE(i, flags, kernel_name, ...)                              \
  if (i == flags) {                                                         \
    kernel_name<T,AccT, i><<<grid, threads, 0, dev_ctx.stream()>>>(__VA_ARGS__); \
  }

// 0 for no scale, no bias
// 1 for has scale, no bias
// 2 for no scale, has bias
// 3 for has scale, has bias
#define UNROLL_ALL_CASES(flags, kernel_name, ...) \
  CHECK_CASE(0, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(1, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(2, flags, kernel_name, __VA_ARGS__)  \
  CHECK_CASE(3, flags, kernel_name, __VA_ARGS__)

template <typename T, typename U>
__global__ void GroupNormForwardGetMeanAndVar(const T* x,
                                              int N,
                                              int C,
                                              int W,
                                              int imsize,
                                              int groups,
                                              int group_size,
                                              U* mean,
                                              U* var) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int number = min(group_size, static_cast<int>(C - gid * group_size));
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  U x_mean = static_cast<U>(0);
  U x_var = static_cast<U>(0);
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    T val;
    int hid = imid / W;
    int wid = imid % W;
    val = x[(bid * H + hid) * W * C + wid * C + ccid];

    x_mean += static_cast<U>(val);
    x_var += static_cast<U>(val * val);
  }
  x_mean /= number * imsize;
  x_var /= number * imsize;
  CudaAtomicAddWithWarp(&mean[bid * groups + gid], x_mean);
  CudaAtomicAddWithWarp(&var[bid * groups + gid], x_var);
}


template <typename T, typename AccT, int flags>
__global__ void GroupNormForward(const T* x,
                                 const AccT* mean,
                                 const AccT* var,
                                 const T* scale,
                                 const T* bias,
                                 int N,
                                 int C,
                                 int W,
                                 int imsize,
                                 int groups,
                                 int group_size,
                                 float epsilon,
                                 T* y,
                                 T* real_mean,
                                 T* real_var,
                                 const DataLayout data_layout) {
  int gid = blockIdx.y;
  int cid = blockIdx.x;
  int bid = blockIdx.z;
  int H = imsize / W;
  int ccid = gid * group_size + cid;
  if (ccid >= C) return;
  auto ng = bid * groups + gid;
  AccT x_mean = static_cast<AccT>(mean[ng]);
  AccT x_var = static_cast<AccT>(var[ng]);
  x_var = x_var - x_mean * x_mean;
  AccT var_inv = rsqrt(x_var + epsilon);
  if (cid == 0 && threadIdx.x == 0) {
    real_var[ng] = x_var;
    real_mean[ng] = x_mean;
  }
  for (int imid = threadIdx.x; imid < imsize; imid += blockDim.x) {
    AccT val;
    int hid, wid;
    int index = (bid * C + ccid) * imsize + imid;
    if (data_layout == DataLayout::kNCHW) {
      val = static_cast<AccT>(x[index]);
    } else {
      hid = imid / W;
      wid = imid % W;
      val = static_cast<AccT>(x[(bid * H + hid) * W * C + wid * C + ccid]);
    }
    val = ((val - x_mean) * var_inv);
    if (flags & kHasScale) {
      val *= static_cast<AccT>(scale[ccid]);
    }
    if (flags & kHasBias) {
      val += static_cast<AccT>(bias[ccid]);
    }
    if (data_layout == DataLayout::kNCHW) {
      y[index] = static_cast<T>(val);
    } else {
      y[(bid * H + hid) * W * C + wid * C + ccid] = static_cast<T>(val);
    }
  }
}

template <typename T, typename Context>
void GroupNormKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const paddle::optional<DenseTensor>& scale,
                     const paddle::optional<DenseTensor>& bias,
                     float epsilon,
                     int groups,
                     const std::string& data_layout_str,
                     DenseTensor* y,
                     DenseTensor* mean,
                     DenseTensor* var) {
  using AccT = typename kps::details::MPTypeTrait<T>::Type;

  const DataLayout data_layout = phi::StringToDataLayout(data_layout_str);
  const auto scale_ptr = scale.get_ptr();
  const auto bias_ptr = bias.get_ptr();

  const auto x_dims = x.dims();
  const int C = (data_layout == DataLayout::kNCHW ? x_dims[1]
                                                  : x_dims[x_dims.size() - 1]);
  const int group_size = C / groups;

  const int W = (data_layout == DataLayout::kNCHW ? x_dims[x_dims.size() - 1]
                                                  : x_dims[x_dims.size() - 2]);

  dev_ctx.template Alloc<T>(y);
  dev_ctx.template Alloc<T>(mean);
  dev_ctx.template Alloc<T>(var);
  phi::funcs::SetConstant<GPUContext, AccT> set_zero;
  DenseTensor temp_var;
  temp_var.Resize(var->dims());
  DenseTensor temp_mean;
  temp_mean.Resize(mean->dims());

  dev_ctx.template Alloc<AccT>(&temp_var);
  dev_ctx.template Alloc<AccT>(&temp_mean);

  auto* x_data = x.data<T>();
  auto* y_data = y->data<T>();
  auto* mean_data = mean->data<T>();
  auto* var_data = var->data<T>();
  auto* temp_var_data = temp_var.data<AccT>();
  auto* temp_mean_data=temp_mean.data<AccT>();


  const T* scale_data = nullptr;
  if (scale_ptr) scale_data = scale_ptr->data<T>();
  const T* bias_data = nullptr;
  if (bias_ptr) bias_data = bias_ptr->data<T>();

  int imsize = 1;
  if (data_layout == DataLayout::kNCHW) {
    for (int i = 2; i < x_dims.size(); ++i) {
      imsize *= x_dims[i];
    }
  } else {
    for (int i = 1; i < x_dims.size() - 1; ++i) {
      imsize *= x_dims[i];
    }
  }

#ifdef __HIPCC__
  int block_size = std::max(std::min(256, imsize), 64);
#else
  int block_size = std::min(1024, imsize);
#endif

  dim3 grid(group_size, groups, x_dims[0]);
  dim3 threads(block_size, 1, 1);
  if (data_layout == DataLayout::kNCHW) {
    constexpr int vec_size = sizeof(float4) / sizeof(T);
    int size = group_size * imsize;
    const int max_num_threads = 1024;
    int max_block_size = std::min(size / vec_size, max_num_threads);
    int block_size_nchw = 1;
    while (block_size_nchw < max_block_size) {
      block_size_nchw *= 2;
    }
    block_size_nchw = std::max(block_size_nchw, kps::details::kWarpSize);
    dim3 grids(x_dims[0] * groups);
    dim3 blocks(block_size_nchw);
    if (size < vec_size * block_size_nchw) {
      ScalarGetMeanAndVarNCHW<T, AccT><<<grids, blocks, 0, dev_ctx.stream()>>>(
          x_data, temp_mean_data, temp_var_data, size);
    } else {
      VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
          <<<grids, blocks, 0, dev_ctx.stream()>>>(
              x_data, temp_mean_data, temp_var_data, size);
    }
  } else {
    set_zero(dev_ctx, &temp_mean, static_cast<AccT>(0));
    set_zero(dev_ctx, &temp_var, static_cast<AccT>(0));
    GroupNormForwardGetMeanAndVar<T, AccT>
        <<<grid, threads, 0, dev_ctx.stream()>>>(x_data,
                                                 x_dims[0],
                                                 C,
                                                 W,
                                                 imsize,
                                                 groups,
                                                 group_size,
                                                 temp_mean_data,
                                                 temp_var_data);
  }
  int flags =
      (scale_data != nullptr) * kHasScale + (bias_data != nullptr) * kHasBias;
  UNROLL_ALL_CASES(flags,
                   GroupNormForward,
                   x_data,
                   temp_mean_data,
                   temp_var_data,
                   scale_data,
                   bias_data,
                   x_dims[0],
                   C,
                   W,
                   imsize,
                   groups,
                   group_size,
                   epsilon,
                   y_data,
                   mean_data,
                   var_data,
                   data_layout);
}

template <typename T, typename AccT>
void GroupNormDirectCUDAFunctor<T,AccT>::operator()(gpuStream_t stream,
                                               const T* input,
                                               std::vector<int> input_shape,
                                               const T* bias,
                                               const T* scale,
                                               AccT* temp_mean,
                                               AccT* temp_variance,
                                               int groups,
                                               float eps,
                                               T* output,
                                               T* mean,
                                               T* variance,
                                               const DataLayout data_layout) {
  const auto input_ddim = phi::make_ddim(input_shape);
  const int C =
      (data_layout == DataLayout::kNCHW ? input_ddim[1]
                                        : input_ddim[input_ddim.size() - 1]);
  const int group_size = C / groups;
  const int W =
      (data_layout == DataLayout::kNCHW ? input_ddim[input_ddim.size() - 1]
                                        : input_ddim[input_ddim.size() - 2]);

  int image_size = 1;
  if (data_layout == DataLayout::kNCHW) {
    for (int i = 2; i < input_ddim.size(); ++i) {
      image_size *= input_ddim[i];
    }
  } else {
    for (int i = 1; i < input_ddim.size() - 1; ++i) {
      image_size *= input_ddim[i];
    }
  }
#ifdef __HIPCC__
  int block_size = std::max(std::min(256, image_size), 64);
#else
  int block_size = std::min(1024, image_size);
#endif
  dim3 grid(group_size, groups, input_ddim[0]);
  dim3 threads(block_size, 1, 1);
  if (data_layout == DataLayout::kNCHW) {
    constexpr int vec_size = sizeof(float4) / sizeof(float);
    int size = group_size * image_size;  // group element size
    const int max_num_threads = 1024;
    int max_block_size = std::min(size / vec_size, max_num_threads);
    int block_size_nchw = 1;
    while (block_size_nchw < max_block_size) {
      block_size_nchw *= 2;
    }

    block_size_nchw = std::max(block_size_nchw, phi::kps::details::kWarpSize);
    dim3 grids(input_ddim[0] * groups);
    dim3 blocks(block_size_nchw);

    if (size < vec_size * block_size_nchw) {
      phi::ScalarGetMeanAndVarNCHW<T,AccT>
          <<<grids, blocks, 0, stream>>>(input, temp_mean, temp_variance, size);
    } else {
      phi::VectorizedGetMeanAndVarNCHW<T, AccT, vec_size>
          <<<grids, blocks, 0, stream>>>(input, temp_mean, temp_variance, size);
    }
  } else {
    
    phi::GroupNormForwardGetMeanAndVar<T,AccT>
        <<<grid, threads, 0, stream>>>(input,
                                       input_ddim[0],
                                       C,
                                       W,
                                       image_size,
                                       groups,
                                       group_size,
                                       temp_mean,
                                       temp_variance);
  }

  GroupNormForward<T, T, 3><<<grid, threads, 0, stream>>>(
      input,
      temp_mean,
      temp_variance,
      scale,
      bias,
      input_ddim[0],
      C,
      W,
      image_size,
      groups,
      group_size,
      eps,
      output,
      mean,
      variance,
      data_layout);  // for now, we only support nchw for group norm
}
template class GroupNormDirectCUDAFunctor<float>;
}  // namespace phi

PD_REGISTER_KERNEL(
    group_norm, GPU, ALL_LAYOUT, phi::GroupNormKernel, float, double,phi::dtype::float16) {}
