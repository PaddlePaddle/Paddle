#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/lerp_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"

#include "paddle/phi/kernels/broadcast_tensors_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/common_shape.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"

namespace phi {

template <typename T>
__global__ void GetLerpGrad(const T* weight,
                            const T* dout,
                            T* dx,
                            T* dy,
                            const int out_size,
                            const int x_size,
                            const int y_size) {
  CUDA_KERNEL_LOOP_TYPE(idx, out_size, int64_t) {
    T temp_dx = weight[idx] * dout[idx];
    if (idx < x_size) {
      dx[idx] = dout[idx] - temp_dx;
    }
    if (idx < y_size) {
      dy[idx] = temp_dx;
    }
  }
}

template <typename T>
__global__ void GetLerpGradRankZero(const T* weight,
                                    const T* dout,
                                    T* dx,
                                    T* dy,
                                    const int out_size,
                                    const int x_size,
                                    const int y_size) {
  CUDA_KERNEL_LOOP_TYPE(idx, out_size, int64_t) {
    T temp_dx = weight[0] * dout[idx];
    if (idx < x_size) {
      dx[idx] = dout[idx] - temp_dx;
    }
    if (idx < y_size) {
      dy[idx] = temp_dx;
    }
  }
}

template <typename T, typename Context, size_t D>
void GetRduceResult(const Context& ctx,
                    const DenseTensor& out_grad,
                    const DenseTensor& b_xgrad,
                    const DenseTensor& b_ygrad,
                    DenseTensor* x_grad,
                    DenseTensor* y_grad) {
  auto& dout = out_grad;
  auto dout_dims = dout.dims();
  auto* dx = x_grad;
  auto* dy = y_grad;
  DDim dx_dims;
  DDim dy_dims;
  Eigen::DSizes<int, D * 2> dx_reshape_dims;
  Eigen::DSizes<int, D * 2> dy_reshape_dims;
  Eigen::DSizes<int, D> reduce_dims;
  Eigen::DSizes<int, D> dx_bcast_dims;
  Eigen::DSizes<int, D> dy_bcast_dims;

  dx_dims = phi::funcs::ExtendDims2Rank(dx->dims(), D);
  phi::funcs::GetBroadcastDims<D>(dx_dims, dout_dims, &dx_bcast_dims);
  dy_dims = phi::funcs::ExtendDims2Rank(dy->dims(), D);
  phi::funcs::GetBroadcastDims<D>(dy_dims, dout_dims, &dy_bcast_dims);
  for (int i = 0; i < dout_dims.size(); ++i) {
    dx_reshape_dims[2 * i] = dx_bcast_dims[i];
    dx_reshape_dims[2 * i + 1] = dx_dims[i];

    dy_reshape_dims[2 * i] = dy_bcast_dims[i];
    dy_reshape_dims[2 * i + 1] = dy_dims[i];
    reduce_dims[i] = 2 * i;
  }

  ctx.template Alloc<T>(dx);
  ctx.template Alloc<T>(dy);
  auto eigen_dx = phi::EigenTensor<T, D>::From(*dx, dx_dims);
  auto eigen_dy = phi::EigenTensor<T, D>::From(*dy, dy_dims);
  dx_dims = phi::funcs::ExtendDims2Rank(x_grad->dims(), D);
  auto broad_dx = phi::EigenTensor<T, D>::From(b_xgrad);
  auto broad_dy = phi::EigenTensor<T, D>::From(b_ygrad);

  auto& place = *ctx.eigen_device();
  eigen_dx.device(place) = broad_dx.reshape(dx_reshape_dims)
                               .sum(reduce_dims)
                               .reshape(eigen_dx.dimensions());
  eigen_dy.device(place) = broad_dy.reshape(dy_reshape_dims)
                               .sum(reduce_dims)
                               .reshape(eigen_dy.dimensions());
}

int XYNeedReduce(const DenseTensor& x,
                 const DenseTensor& y,
                 const DenseTensor& out) {
  // 不考虑不可broadcast的情况，在算子调用时已排除
  auto x_dims = x.dims();
  auto y_dims = y.dims();
  auto out_dims = out.dims();
  int x_rank = x_dims.size();
  int y_rank = y_dims.size();
  int out_rank = out_dims.size();
  int smaller_rank = std::min(x_rank, y_rank);
  if (std::max(x_rank, y_rank) < out_rank) {
    return 1;
  }
  for (int i = 1; i <= smaller_rank; ++i) {
    int x_idx = x_rank - i;
    int y_idx = y_rank - i;
    int out_idx = out_rank - i;
    if (x_dims[x_idx] != y_dims[y_idx]) {
      return 1;
    }
    if (x_dims[x_idx] == 1 && y_dims[y_idx] == 1 && out_dims[out_idx] != 1) {
      return 1;
    }
  }
  return 0;
}

template <typename T, typename Context>
void SwitchKernel(const Context& ctx,
                  const DenseTensor& weight,
                  const DenseTensor& out_grad,
                  const int x_grad_size,
                  const int y_grad_size,
                  T* x_grad_data,
                  T* y_grad_data) {
  if (weight.dims().size() == 1) {
    const T* weight_data = weight.data<T>();
    const T* out_grad_data = out_grad.data<T>();
    const int out_size = out_grad.numel();
    const int weight_size = weight.numel();
    auto gpu_config = phi::backends::gpu::GetGpuLaunchConfig1D(ctx, out_size);
    GetLerpGradRankZero<T><<<gpu_config.GetGridSize(),
                             gpu_config.GetBlockSize(),
                             0,
                             ctx.stream()>>>(weight_data,
                                             out_grad_data,
                                             x_grad_data,
                                             y_grad_data,
                                             out_size,
                                             x_grad_size,
                                             y_grad_size);
  } else {
    // 首先对weight进行braodcast，使用
    // phi::BroadcastTensorsKernel，使其维度和out_grad一致
    const std::vector<const DenseTensor*> in_tensors = {&weight, &out_grad};
    DenseTensor b_weight = phi::EmptyLike<T>(ctx, out_grad);
    DenseTensor b_out = phi::EmptyLike<T>(ctx, out_grad);
    std::vector<DenseTensor*> out_tensors = {&b_weight, &b_out};

    phi::BroadcastTensorsKernel<T, Context>(ctx, in_tensors, out_tensors);

    const T* weight_data = b_weight.data<T>();
    const T* out_grad_data = b_out.data<T>();
    const int out_size = out_grad.numel();
    const int weight_size = weight.numel();

    auto gpu_config = phi::backends::gpu::GetGpuLaunchConfig1D(ctx, out_size);
    GetLerpGrad<T><<<gpu_config.GetGridSize(),
                     gpu_config.GetBlockSize(),
                     0,
                     ctx.stream()>>>(weight_data,
                                     out_grad_data,
                                     x_grad_data,
                                     y_grad_data,
                                     out_size,
                                     x_grad_size,
                                     y_grad_size);
  }
}

template <typename T, typename Context>
void LerpGradKernel(const Context& ctx,
                    const DenseTensor& x,
                    const DenseTensor& y,
                    const DenseTensor& weight,
                    const DenseTensor& out,
                    const DenseTensor& out_grad,
                    DenseTensor* x_grad,
                    DenseTensor* y_grad) {
  const int rank = out.dims().size();
  PADDLE_ENFORCE_GE(
      rank,
      1,
      phi::errors::InvalidArgument(
          "The number of dimensions for LerpGradOp must be "
          "greater than or equal to 1, but the value received is %d.",
          rank));
  PADDLE_ENFORCE_LE(
      rank,
      6,
      phi::errors::InvalidArgument(
          "The number of dimensions for LerpGradOp must be "
          "less than or equal to 6, but the value received is %d.",
          rank));

  // 判断x_grad, y_grad
  // 是否需要reduce，需要reduce的话就先进行broadcast用b_xgrad,
  // b_ygrad。不需要的话就用x_grad, y_grad。
  // 如果x,y在中间有某个维度不一致，或者和weight的中间某个维度不一致，就需要先broadcast再reduce。
  //  例如 case1:  x:2*1*3, y:2*2*3  w:2*2*3 => out: 2*2*3
  //      case2:  x:2*1:3, y: 2*1*3 w:2*2*3 => out: 2*2*3
  // 如果x,y在初始维度不一致，就无所谓，只要控制kernel写入的idx大小不越界访问内存就可以。例如
  // x:1*2*3, y:2*2*3，无需reduce。

  int reduce_flag = XYNeedReduce(x, y, out);
  if (reduce_flag == 0) {
    T* x_grad_data = ctx.template Alloc<T>(x_grad);
    T* y_grad_data = ctx.template Alloc<T>(y_grad);
    int x_grad_size = x.numel();
    int y_grad_size = y.numel();

    SwitchKernel<T, Context>(ctx,
                             weight,
                             out_grad,
                             x_grad_size,
                             y_grad_size,
                             x_grad_data,
                             y_grad_data);

  } else {
    DenseTensor b_xgrad = phi::EmptyLike<T, Context>(ctx, out_grad);
    DenseTensor b_ygrad = phi::EmptyLike<T, Context>(ctx, out_grad);
    T* x_grad_data = ctx.template Alloc<T>(&b_xgrad);
    T* y_grad_data = ctx.template Alloc<T>(&b_ygrad);
    int x_grad_size = out.numel();
    int y_grad_size = out.numel();

    SwitchKernel<T, Context>(ctx,
                             weight,
                             out_grad,
                             x_grad_size,
                             y_grad_size,
                             x_grad_data,
                             y_grad_data);

    switch (rank) {
      case 1:
        GetRduceResult<T, Context, 1>(
            ctx, out_grad, b_xgrad, b_ygrad, x_grad, y_grad);
        break;
      case 2:
        GetRduceResult<T, Context, 2>(
            ctx, out_grad, b_xgrad, b_ygrad, x_grad, y_grad);
        break;
      case 3:
        GetRduceResult<T, Context, 3>(
            ctx, out_grad, b_xgrad, b_ygrad, x_grad, y_grad);
        break;
      case 4:
        GetRduceResult<T, Context, 4>(
            ctx, out_grad, b_xgrad, b_ygrad, x_grad, y_grad);
        break;
      case 5:
        GetRduceResult<T, Context, 5>(
            ctx, out_grad, b_xgrad, b_ygrad, x_grad, y_grad);
        break;
      case 6:
        GetRduceResult<T, Context, 6>(
            ctx, out_grad, b_xgrad, b_ygrad, x_grad, y_grad);
        break;
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(
    lerp_grad, GPU, ALL_LAYOUT, phi::LerpGradKernel, float, double) {}
