#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/cross_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"

namespace phi {

template <typename T>
__global__ void CrossGrad(const T* x,
                          const T* y,
                          const T* out,
                          T* out_dx,
                          T* out_dy,
                          const int stride,
                          const int N,
                          phi::funcs::IndexCalculator index_calculator) {
  CUDA_KERNEL_LOOP(i, N) {
    int offset = index_calculator(i);

    auto pos0 = offset + 0 * stride;
    auto pos1 = offset + 1 * stride;
    auto pos2 = offset + 2 * stride;

    out_dx[pos0] = out[pos2] * y[pos1] - out[pos1] * y[pos2];
    out_dy[pos0] = out[pos1] * x[pos2] - out[pos2] * x[pos1];

    out_dx[pos1] = out[pos0] * y[pos2] - out[pos2] * y[pos0];
    out_dy[pos1] = out[pos2] * x[pos0] - out[pos0] * x[pos2];

    out_dx[pos2] = out[pos1] * y[pos0] - out[pos0] * y[pos1];
    out_dy[pos2] = out[pos0] * x[pos1] - out[pos1] * x[pos0];
  }
}

template <typename T, typename Context>
void CrossGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     const DenseTensor& out_grad,
                     int axis,
                     DenseTensor* x_grad,
                     DenseTensor* y_grad) {
  auto& input_x = x;
  auto& input_y = y;
  auto& input_out_grad = out_grad;
  auto* output_x_grad = x_grad;
  auto* output_y_grad = y_grad;
  int dim = axis;

  auto input_x_dims = input_x.dims();
  if (dim != DDim::kMaxRank) {
    PADDLE_ENFORCE_EQ(
        dim < input_x_dims.size() && dim >= (0 - input_x_dims.size()),
        true,
        errors::OutOfRange(
            "Attr(dim) is out of range, It's expected "
            "to be in range of [-%d, %d]. But received Attr(dim) = %d.",
            input_x_dims.size(),
            input_x_dims.size() - 1,
            dim));
    if (dim < 0) {
      dim += input_x_dims.size();
    }

    PADDLE_ENFORCE_EQ(
        input_x_dims[dim] == 3,
        true,
        errors::InvalidArgument(
            "Input(X/Y).dims[dim] must be equal to 3. But received: "
            "Input(X/Y).dims[dim] = [%d].",
            input_x_dims[dim]));
  } else {
    for (auto i = 0; i < input_x_dims.size(); i++) {
      if (input_x_dims[i] == 3) {
        dim = i;
        break;
      }
    }
    PADDLE_ENFORCE_EQ(
        dim == DDim::kMaxRank,
        false,
        errors::InvalidArgument("There must be at least one dimension 'd' "
                                "so that Input(X/Y).dims()[d] is equal to 3. "
                                "But received: Input(X/Y).dims() == [%s].",
                                input_x_dims));
  }

  std::vector<int> cal_dims;
  std::vector<int> left_strides;
  std::vector<int> full_strides;
  std::vector<int> merged_dims;

  for (int i = 0; i < dim; i++) {
    if (i == 0) {
      merged_dims.push_back(input_x_dims[i]);
    } else {
      merged_dims[0] *= input_x_dims[i];
    }
  }
  int merge_axis = merged_dims.size();
  merged_dims.push_back(input_x_dims[dim]);
  for (int i = dim + 1; i < input_x_dims.size(); i++) {
    if (i == dim + 1) {
      merged_dims.push_back(input_x_dims[i]);
    } else {
      merged_dims[merge_axis + 1] *= input_x_dims[i];
    }
  }

  int full_dim = 1;
  for (int i = 0; i < merged_dims.size(); i++) {
    full_strides.insert(full_strides.begin(), full_dim);
    full_dim *= merged_dims[merged_dims.size() - i - 1];
    if (i == merge_axis) {
      continue;
    }
    cal_dims.push_back(i);
  }
  int left_dim = 1;
  for (int i = merged_dims.size() - 1; i >= 0; i--) {
    if (i == merge_axis) {
      continue;
    }
    left_strides.insert(left_strides.begin(), left_dim);
    left_dim *= merged_dims[i];
  }

  const auto* input_x_data = input_x.data<T>();
  const auto* input_y_data = input_y.data<T>();
  const auto* input_out_grad_data = input_out_grad.data<T>();
  auto* output_x_grad_data = dev_ctx.template Alloc<T>(x_grad);
  auto* output_y_grad_data = dev_ctx.template Alloc<T>(y_grad);
  auto index_calculator = phi::funcs::IndexCalculator(
      merged_dims.size() - 1, cal_dims, left_strides, full_strides);

  int64_t numel = x.numel();
  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel / 3);

  CrossGrad<<<config.block_per_grid,
              config.thread_per_block,
              0,
              dev_ctx.stream()>>>(input_x_data,
                                  input_y_data,
                                  input_out_grad_data,
                                  output_x_grad_data,
                                  output_y_grad_data,
                                  full_strides[merge_axis],
                                  numel / 3,
                                  index_calculator);
}
}  // namespace phi

PD_REGISTER_KERNEL(cross_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CrossGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
