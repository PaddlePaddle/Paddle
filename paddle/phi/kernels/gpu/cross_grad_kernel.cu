#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cross_grad_kernel.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename T, typename IndexCalculator>
__global__ void CrossGrad(const T* x,
                          const T* y,
                          const T* out,
                          T* out_dx,
                          T* out_dy,
                          const int stride,
                          const int N,
                          IndexCalculator index_calculator) {
  CUDA_KERNEL_LOOP(i, N) {
    int offset = index_calculator(i);

    auto pos0 = offset + 0 * stride;
    auto pos1 = offset + 1 * stride;
    auto pos2 = offset + 2 * stride;

    out_dx[pos0] = out[pos2] * y[pos1] - out[pos1] * y[pos2];
    out_dy[pos0] = out[pos1] * x[pos2] - out[pos2] * x[pos1];

    out_dx[pos1] = out[pos0] * y[pos2] - out[pos2] * y[pos0];
    out_dy[pos1] = out[pos2] * x[pos0] - out[pos0] * x[pos2];

    out_dx[pos2] = out[pos1] * y[pos0] - out[pos0] * y[pos1];
    out_dy[pos2] = out[pos0] * x[pos1] - out[pos1] * x[pos0];
  }
}

template <typename T, typename Context>
void CrossGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& y,
                     const DenseTensor& out_grad,
                     int axis,
                     DenseTensor* x_grad,
                     DenseTensor* y_grad) {
  auto& input_x = x;
  auto& input_y = y;
  auto& input_out_grad = out_grad;
  auto* output_x_grad = x_grad;
  auto* output_y_grad = y_grad;
  int dim = axis;

  auto input_x_dims = input_x.dims();
  if (dim != DDim::kMaxRank) {
    PADDLE_ENFORCE_EQ(
        dim < input_x_dims.size() && dim >= (0 - input_x_dims.size()),
        true,
        errors::OutOfRange(
            "Attr(dim) is out of range, It's expected "
            "to be in range of [-%d, %d]. But received Attr(dim) = %d.",
            input_x_dims.size(),
            input_x_dims.size() - 1,
            dim));
    if (dim < 0) {
      dim += input_x_dims.size();
    }

    PADDLE_ENFORCE_EQ(
        input_x_dims[dim] == 3,
        true,
        errors::InvalidArgument(
            "Input(X/Y).dims[dim] must be equal to 3. But received: "
            "Input(X/Y).dims[dim] = [%d].",
            input_x_dims[dim]));
  } else {
    for (auto i = 0; i < input_x_dims.size(); i++) {
      if (input_x_dims[i] == 3) {
        dim = i;
        break;
      }
    }
    PADDLE_ENFORCE_EQ(
        dim == DDim::kMaxRank,
        false,
        errors::InvalidArgument("There must be at least one dimension 'd' "
                                "so that Input(X/Y).dims()[d] is equal to 3. "
                                "But received: Input(X/Y).dims() == [%s].",
                                input_x_dims));
  }

  std::vector<int> cal_dims;
  std::vector<int> left_strides;
  std::vector<int> full_strides;

  int full_dim = 1;
  int left_dim = 1;
  for (auto i = 0; i < input_x_dims.size(); i++) {
    full_strides.insert(full_strides.begin(), full_dim);
    full_dim *= input_x_dims[input_x_dims.size() - i - 1];
    if (i == dim) {
      continue;
    }
    cal_dims.push_back(i);
    left_strides.insert(left_strides.begin(), left_dim);
    left_dim *= input_x_dims[input_x_dims.size() - i - 1];
  }

  const auto* input_x_data = input_x.data<T>();
  const auto* input_y_data = input_y.data<T>();
  const auto* input_out_grad_data = input_out_grad.data<T>();

  auto* output_x_grad_data = dev_ctx.template Alloc<T>(x_grad);
  auto* output_y_grad_data = dev_ctx.template Alloc<T>(y_grad);

  auto index_calculator = phi::funcs::IndexCalculator(
      input_x_dims.size() - 1, cal_dims, left_strides, full_strides);

  int64_t numel = x.numel();
  int threads = PADDLE_CUDA_NUM_THREADS;
  int blocks = (numel + threads - 1) / threads;

  CrossGrad<<<blocks, threads>>>(input_x_data,
                                 input_y_data,
                                 input_out_grad_data,
                                 output_x_grad_data,
                                 output_y_grad_data,
                                 full_strides[dim],
                                 numel / 3,
                                 index_calculator);
}
}  // namespace phi

PD_REGISTER_KERNEL(cross_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CrossGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
