#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/triu_indices_kernel.h"

#include <algorithm>
#include <tuple>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T>
__device__ inline int resolve_root_int(int b, int cX4, int x, int32_t sign) {
  int64_t bXb_cX4 = b * b - cX4;
  double sr = ::sqrt(static_cast<double>(bXb_cX4));
  T res = ::__double2ll_rd((-b + sign * sr) / 2);
  if (bXb_cX4 != static_cast<int>(sr * sr)) {
    int llsr = ::__double2ll_rd(sr);
    int diff = ::__double2ll_ru(
        ::sqrt(::fabs(static_cast<double>(bXb_cX4 - llsr * llsr))));
    auto l = res > diff ? res - diff : 0;
    auto r = res + diff + 1;
    x <<= 1;
    while (l + 1 < r) {
      auto m = (l + r) >> 1;
      if (sign * (b + m) * m > x) {
        r = m;
      } else {
        l = m;
      }
    }
    res = l;
  }
  return res;
}

template <typename T>
__device__ inline void get_coordinate_in_triu_trapezoid(int f,
                                                        int x,
                                                        T* row,
                                                        T* col) {
  f <<= 1;  // all statements use 2f, so only calculate it once here.
  auto b = -1 - f;
  auto cX4 = x << 3;  // 4 * c = 4 * (2x) = 8x;
  *row = resolve_root_int<T>(b, cX4, x, -1);
  *col = (x - (((f - *row + 1) * *row) >> 1)) + *row;
}

template <typename T>
__global__ void triu_indices_kernel(T* out_data,
                                    int col_offset,
                                    int m_first_row,
                                    int col,
                                    int rectangle_size,
                                    int triu_size) {
  int linear_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (linear_index < triu_size) {
    T r, c;
    if (linear_index < rectangle_size) {
      // the coordinate is within the top rectangle
      r = linear_index / col;
      c = linear_index % col;
    } else {
      // the coordinate falls in the bottom trapezoid
      get_coordinate_in_triu_trapezoid<T>(
          m_first_row, linear_index - rectangle_size, &r, &c);
      r += rectangle_size / col;
    }

    c += col_offset;
    out_data[linear_index] = r;
    out_data[linear_index + triu_size] = c;
  }
}

template <typename T, typename Context>
void TriuIndicesKernel(const Context& dev_ctx,
                       int row,
                       int col,
                       int offset,
                       DataType dtype,
                       DenseTensor* out) {
  T* out_data = dev_ctx.template Alloc<T>(out);
  auto out_dims = out->dims();
  int triu_size = out_dims[1];
  //  auto tensor = empty_cuda({2, triu_size}, dtype_opt, layout_opt,
  //  device_opt, pin_memory_opt);

  if (triu_size > 0) {
    // # of triu elements in the first row
    auto m_first_row = offset > 0 ? std::max<int>(col - offset, 0)
                                  :  // upper bounded by col
                           col;

    // size of the top rectangle
    int rectangle_size = 0;
    if (offset < 0) {
      rectangle_size = std::min<int>(row, -offset) * col;
    }

    //  using gpu_launch_config to get grid_size and block_size
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, triu_size);

    triu_indices_kernel<T><<<config.block_per_grid.x,
                             config.thread_per_block.x,
                             0,
                             dev_ctx.stream()>>>(out_data,
                                                 std::max<int>(0, offset),
                                                 m_first_row,
                                                 col,
                                                 rectangle_size,
                                                 triu_size);
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(
    triu_indices, GPU, ALL_LAYOUT, phi::TriuIndicesKernel, int, int64_t) {}
