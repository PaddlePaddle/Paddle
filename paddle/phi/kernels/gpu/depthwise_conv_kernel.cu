// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/conv_op.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cpu/conv_util.h"
#include "paddle/phi/kernels/funcs/batch_norm_utils.h"
#include "paddle/phi/kernels/gpu/depthwise_conv.h"

namespace phi {

template <typename T, typename Context>
void DepthwiseConvKernel(const Context& dev_ctx,
                         const DenseTensor& input,
                         const DenseTensor& filter,
                         const std::vector<int>& strides_t,
                         const std::vector<int>& paddings_t,
                         const std::string& padding_algorithm,
                         int groups,
                         const std::vector<int>& dilations_t,
                         const std::string& data_format,
                         bool use_addto,
                         int workspace_size_MB,
                         bool exhaustive_search,
                         bool fuse_relu,
                         DenseTensor* out) {
  DenseTensor* output = out;
  output->mutable_data<T>(dev_ctx.GetPlace());

  const std::vector<int> strides = strides_t;
  std::vector<int> dilations = dilations_t;
  std::vector<int> paddings = paddings_t;

  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");
  if (channel_last) {
    PADDLE_ENFORCE_EQ(
        output->dims()[output->dims().size() - 1] %
            input.dims()[input.dims().size() - 1],
        0,
        phi::errors::InvalidArgument(
            "ShapeError: The output channels must be a multiple of the "
            "input channels. But receivced output channel number is %d "
            "and input channel number is %d",
            output->dims()[output->dims().size() - 1],
            input.dims()[input.dims().size() - 1]));
  } else {
    PADDLE_ENFORCE_EQ(
        output->dims()[1] % input.dims()[1],
        0,
        phi::errors::InvalidArgument(
            "ShapeError: The output channels must be a multiple of the "
            "input channels. But receivced output channel number is %d "
            "and input channel number is %d",
            output->dims()[1],
            input.dims()[1]));
  }

  // update padding and dilation
  auto in_dims = input.dims();
  auto filter_dims = filter.dims();

  DDim in_data_dims;
  const paddle::framework::DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_format);
  if (data_layout != paddle::framework::DataLayout::kNHWC) {
    in_data_dims = slice_ddim(in_dims, 2, in_dims.size());
  } else {
    in_data_dims = slice_ddim(in_dims, 1, in_dims.size() - 1);
  }

  DDim filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings, &dilations, padding_algorithm, in_data_dims, strides, ksize);

  bool is_sys_pad = strides.size() * 2 == paddings.size() ? false : true;
  if (!is_sys_pad) {
    for (size_t i = 0; i < strides.size(); ++i) {
      paddings.erase(paddings.begin() + i + 1);
    }
  }

  if (fuse_relu) {
    paddle::operators::math::DepthwiseConvFunctor<Context, T, true>
        depthwiseConv;
    depthwiseConv(dev_ctx,
                  input,
                  filter,
                  strides,
                  paddings,
                  dilations,
                  output,
                  data_layout);
  } else {
    paddle::operators::math::DepthwiseConvFunctor<Context, T, false>
        depthwiseConv;
    depthwiseConv(dev_ctx,
                  input,
                  filter,
                  strides,
                  paddings,
                  dilations,
                  output,
                  data_layout);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(depthwise_conv2d,
                   GPU,
                   ALL_LAYOUT,
                   phi::DepthwiseConvKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
