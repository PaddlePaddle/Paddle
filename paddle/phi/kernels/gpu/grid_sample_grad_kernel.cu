#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/grid_sample_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/grid_sample_utils.h"

namespace phi {

template <typename T>
static __forceinline__ __device__ void AtomicAdd(
    T* data, int h, int w, int sH, int sW, int H, int W, T delta) {
  if (InBounds(h, w, H, W)) {
    paddle::platform::CudaAtomicAdd(data + h * sH + w * sW, delta);
  }
}

template <typename T>
static __forceinline__ __device__ void AtomicAdd3D(T* data,
                                                   int d,
                                                   int h,
                                                   int w,
                                                   int sD,
                                                   int sH,
                                                   int sW,
                                                   int D,
                                                   int H,
                                                   int W,
                                                   T delta) {
  if (InBounds3D(d, h, w, D, H, W)) {
    paddle::platform::CudaAtomicAdd(data + d * sD + h * sH + w * sW, delta);
  }
}

template <typename T>
static __forceinline__ __device__ T
UnnormalizeWithMask(T coord, int size, bool align_corners, T* grad_in) {
  if (align_corners) {
    *grad_in = static_cast<T>(size - 1) / 2;
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    *grad_in = static_cast<T>(size) / 2;
    return ((coord + 1.f) * size - 1) / 2;
  }
}

template <typename T>
static __forceinline__ __device__ T ClipIndexesWithMask(T in,
                                                        int clip_limit,
                                                        T* grad_in) {
  if (in <= static_cast<T>(0)) {
    *grad_in = static_cast<T>(0);
    return static_cast<T>(0);
  } else {
    T max = static_cast<T>(clip_limit - 1);
    if (in >= max) {
      *grad_in = static_cast<T>(0);
      return max;
    } else {
      *grad_in = static_cast<T>(1);
      return in;
    }
  }
}

template <typename T>
static __forceinline__ __device__ T
ReflectIndexesWithMask(T in, int twice_low, int twice_high, T* grad_in) {
  if (twice_low == twice_high) {
    *grad_in = static_cast<T>(0);
    return static_cast<T>(0);
  }
  int grad_in_mult_;
  T min = static_cast<T>(twice_low) / 2;
  T span = static_cast<T>(twice_high - twice_low) / 2;
  in = in - min;
  if (in < static_cast<T>(0)) {
    grad_in_mult_ = -1;
    in = -in;
  } else {
    grad_in_mult_ = 1;
  }
  T extra = fmod(in, span);
  int flips = static_cast<int>(floor(in / span));
  if (flips % 2 == 0) {
    *grad_in = static_cast<T>(grad_in_mult_);
    return extra + min;
  } else {
    *grad_in = static_cast<T>(-grad_in_mult_);
    return span - extra + min;
  }
}

template <typename T>
static __forceinline__ __device__ T
ComputePositionsWithMask(T coord,
                         int size,
                         PaddingMode padding_mode,
                         bool align_corners,
                         T* grad_in) {
  T grad_clip, grad_refl;
  coord = UnnormalizeWithMask<T>(coord, size, align_corners, grad_in);
  if (padding_mode == PaddingMode::border) {
    coord = ClipIndexesWithMask(coord, size, &grad_clip);
    *grad_in = (*grad_in) * grad_clip;
  } else if (padding_mode == PaddingMode::reflect) {
    if (align_corners) {
      coord = ReflectIndexesWithMask(coord, 0, 2 * (size - 1), &grad_refl);
    } else {
      coord = ReflectIndexesWithMask(coord, -1, 2 * size - 1, &grad_refl);
    }
    coord = ClipIndexesWithMask(coord, size, &grad_clip);
    *grad_in = (*grad_in) * grad_refl * grad_clip;
  }

  return coord;
}

template <typename T>
__global__ void GridSamplerCudaBackwardKernel(const int nthreads,
                                              const T* grad_output,
                                              const T* input,
                                              const T* grid,
                                              int n,
                                              int out_c,
                                              int out_h,
                                              int out_w,
                                              int in_h,
                                              int in_w,
                                              T* grad_input,
                                              T* grad_grid,
                                              const Mode mode,
                                              const PaddingMode padding_mode,
                                              bool align_corners) {
  int inp_sN = out_c * in_h * in_w;
  int inp_sC = in_h * in_w;
  int inp_sH = in_w;
  int inp_sW = 1;
  int grid_sN = out_h * out_w * 2;
  int grid_sH = out_w * 2;
  int grid_sW = 2;
  int grid_sCoor = 1;

  int gOut_sN = out_c * out_h * out_w;
  int gOut_sC = out_h * out_w;
  int gOut_sH = out_w;
  int gOut_sW = 1;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_w;
    const int h = (index / out_w) % out_h;
    const int n = index / (out_h * out_w);
    const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];

    T gix_mult, giy_mult;
    ix = ComputePositionsWithMask(
        ix, in_w, padding_mode, align_corners, &gix_mult);
    iy = ComputePositionsWithMask(
        iy, in_h, padding_mode, align_corners, &giy_mult);

    if (mode == Mode::bilinear) {
      int ix_nw = static_cast<int>(floor(ix));
      int iy_nw = static_cast<int>(floor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      T nw = (ix_se - ix) * (iy_se - iy);
      T ne = (ix - ix_sw) * (iy_sw - iy);
      T sw = (ix_ne - ix) * (iy - iy_ne);
      T se = (ix - ix_nw) * (iy - iy_nw);

      T gix = static_cast<T>(0), giy = static_cast<T>(0);
      int gOut_offset = n * gOut_sN + h * gOut_sH + w * gOut_sW;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      int inp_offset_NC = n * inp_sN;
      for (int c = 0; c < out_c; ++c,
               inp_offset_NC += inp_sC,
               gInp_ptr_NC += inp_sC,
               gOut_offset += gOut_sC) {
        T gOut = grad_output[gOut_offset];

        AtomicAdd(
            gInp_ptr_NC, iy_nw, ix_nw, inp_sH, inp_sW, in_h, in_w, nw * gOut);
        AtomicAdd(
            gInp_ptr_NC, iy_ne, ix_ne, inp_sH, inp_sW, in_h, in_w, ne * gOut);
        AtomicAdd(
            gInp_ptr_NC, iy_sw, ix_sw, inp_sH, inp_sW, in_h, in_w, sw * gOut);
        AtomicAdd(
            gInp_ptr_NC, iy_se, ix_se, inp_sH, inp_sW, in_h, in_w, se * gOut);

        if (InBounds(iy_nw, ix_nw, in_h, in_w)) {
          T nw_val = input[inp_offset_NC + iy_nw * inp_sH + ix_nw * inp_sW];
          gix -= nw_val * (iy_se - iy) * gOut;
          giy -= nw_val * (ix_se - ix) * gOut;
        }
        if (InBounds(iy_ne, ix_ne, in_h, in_w)) {
          T ne_val = input[inp_offset_NC + iy_ne * inp_sH + ix_ne * inp_sW];
          gix += ne_val * (iy_sw - iy) * gOut;
          giy -= ne_val * (ix - ix_sw) * gOut;
        }
        if (InBounds(iy_sw, ix_sw, in_h, in_w)) {
          T sw_val = input[inp_offset_NC + iy_sw * inp_sH + ix_sw * inp_sW];
          gix -= sw_val * (iy - iy_ne) * gOut;
          giy += sw_val * (ix_ne - ix) * gOut;
        }
        if (InBounds(iy_se, ix_se, in_h, in_w)) {
          T se_val = input[inp_offset_NC + iy_se * inp_sH + ix_se * inp_sW];
          gix += se_val * (iy - iy_nw) * gOut;
          giy += se_val * (ix - ix_nw) * gOut;
        }
      }

      if (grad_grid != nullptr) {
        T* gGrid_ptr_NHW = grad_grid + index * grid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      }
    } else if (mode == Mode::nearest) {
      int ix_nearest = static_cast<int>(std::nearbyint(ix));
      int iy_nearest = static_cast<int>(std::nearbyint(iy));

      int gOut_offset = n * gOut_sN + h * gOut_sH + w * gOut_sW;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      for (int c = 0; c < out_c;
           ++c, gInp_ptr_NC += inp_sC, gOut_offset += gOut_sC) {
        AtomicAdd(gInp_ptr_NC,
                  iy_nearest,
                  ix_nearest,
                  inp_sH,
                  inp_sW,
                  in_h,
                  in_w,
                  grad_output[gOut_offset]);
      }

      if (grad_grid != nullptr) {
        T* gGrid_ptr_NHW = grad_grid + index * grid_sW;
        gGrid_ptr_NHW[0] = static_cast<T>(0);
        gGrid_ptr_NHW[1] = static_cast<T>(0);
      }
    }
  }
}

template <typename T>
__global__ void GridSampler3DCudaBackwardKernel(const int nthreads,
                                                const T* grad_output,
                                                const T* input,
                                                const T* grid,
                                                int out_c,
                                                int out_d,
                                                int out_h,
                                                int out_w,
                                                int in_d,
                                                int in_h,
                                                int in_w,
                                                T* grad_input,
                                                T* grad_grid,
                                                const Mode mode,
                                                const PaddingMode padding_mode,
                                                bool align_corners) {
  int inp_sW = 1;
  int inp_sH = in_w;
  int inp_sD = in_h * in_w;
  int inp_sC = in_d * inp_sD;
  int inp_sN = out_c * inp_sC;

  int grid_sCoor = 1;
  int grid_sW = 3;
  int grid_sH = out_w * grid_sW;
  int grid_sD = out_h * grid_sH;
  int grid_sN = out_d * grid_sD;

  int gOut_sW = 1;
  int gOut_sH = out_w;
  int gOut_sD = out_h * out_w;
  int gOut_sC = out_d * gOut_sD;
  int gOut_sN = out_c * gOut_sC;

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, int) {
    const int w = index % out_w;
    const int h = (index / out_w) % out_h;
    const int d = (index / (out_h * out_w)) % out_d;
    const int n = index / (out_d * out_h * out_w);
    const auto grid_offset =
        n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

    // get the corresponding input x, y, z co-ordinates from grid
    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];
    T iz = grid[grid_offset + 2 * grid_sCoor];

    // multipliers for gradients on ix, iy, and iz
    T gix_mult, giy_mult, giz_mult;
    ix = ComputePositionsWithMask(
        ix, in_w, padding_mode, align_corners, &gix_mult);
    iy = ComputePositionsWithMask(
        iy, in_h, padding_mode, align_corners, &giy_mult);
    iz = ComputePositionsWithMask(
        iz, in_d, padding_mode, align_corners, &giz_mult);

    if (mode == Mode::bilinear) {
      // get corner pixel values from (x, y, z)
      // for 4d, we used north-east-south-west
      // for 5d, we add top-bottom
      int ix_tnw = static_cast<int>(std::floor(ix));
      int iy_tnw = static_cast<int>(std::floor(iy));
      int iz_tnw = static_cast<int>(std::floor(iz));

      int ix_tne = ix_tnw + 1;
      int iy_tne = iy_tnw;
      int iz_tne = iz_tnw;

      int ix_tsw = ix_tnw;
      int iy_tsw = iy_tnw + 1;
      int iz_tsw = iz_tnw;

      int ix_tse = ix_tnw + 1;
      int iy_tse = iy_tnw + 1;
      int iz_tse = iz_tnw;

      int ix_bnw = ix_tnw;
      int iy_bnw = iy_tnw;
      int iz_bnw = iz_tnw + 1;

      int ix_bne = ix_tnw + 1;
      int iy_bne = iy_tnw;
      int iz_bne = iz_tnw + 1;

      int ix_bsw = ix_tnw;
      int iy_bsw = iy_tnw + 1;
      int iz_bsw = iz_tnw + 1;

      int ix_bse = ix_tnw + 1;
      int iy_bse = iy_tnw + 1;
      int iz_bse = iz_tnw + 1;

      // get surfaces to each neighbor:
      T tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
      T tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
      T tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
      T tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
      T bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
      T bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
      T bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
      T bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

      T gix = static_cast<T>(0), giy = static_cast<T>(0),
        giz = static_cast<T>(0);
      int gOut_offset = n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
      int inp_offset_NC = n * inp_sN;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      for (int c = 0; c < out_c; ++c,
               gOut_offset += gOut_sC,
               gInp_ptr_NC += inp_sC,
               inp_offset_NC += inp_sC) {
        T gOut = grad_output[gOut_offset];

        AtomicAdd3D(gInp_ptr_NC,
                    iz_tnw,
                    iy_tnw,
                    ix_tnw,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    tnw * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_tne,
                    iy_tne,
                    ix_tne,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    tne * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_tsw,
                    iy_tsw,
                    ix_tsw,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    tsw * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_tse,
                    iy_tse,
                    ix_tse,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    tse * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_bnw,
                    iy_bnw,
                    ix_bnw,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    bnw * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_bne,
                    iy_bne,
                    ix_bne,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    bne * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_bsw,
                    iy_bsw,
                    ix_bsw,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    bsw * gOut);
        AtomicAdd3D(gInp_ptr_NC,
                    iz_bse,
                    iy_bse,
                    ix_bse,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    bse * gOut);

        // calculate grad_grid
        if (InBounds3D(iz_tnw, iy_tnw, ix_tnw, in_d, in_h, in_w)) {
          T tnw_val = input[inp_offset_NC + iz_tnw * inp_sD + iy_tnw * inp_sH +
                            ix_tnw * inp_sW];
          gix -= tnw_val * (iy_bse - iy) * (iz_bse - iz) * gOut;
          giy -= tnw_val * (ix_bse - ix) * (iz_bse - iz) * gOut;
          giz -= tnw_val * (ix_bse - ix) * (iy_bse - iy) * gOut;
        }
        if (InBounds3D(iz_tne, iy_tne, ix_tne, in_d, in_h, in_w)) {
          T tne_val = input[inp_offset_NC + iz_tne * inp_sD + iy_tne * inp_sH +
                            ix_tne * inp_sW];
          gix += tne_val * (iy_bsw - iy) * (iz_bsw - iz) * gOut;
          giy -= tne_val * (ix - ix_bsw) * (iz_bsw - iz) * gOut;
          giz -= tne_val * (ix - ix_bsw) * (iy_bsw - iy) * gOut;
        }
        if (InBounds3D(iz_tsw, iy_tsw, ix_tsw, in_d, in_h, in_w)) {
          T tsw_val = input[inp_offset_NC + iz_tsw * inp_sD + iy_tsw * inp_sH +
                            ix_tsw * inp_sW];
          gix -= tsw_val * (iy - iy_bne) * (iz_bne - iz) * gOut;
          giy += tsw_val * (ix_bne - ix) * (iz_bne - iz) * gOut;
          giz -= tsw_val * (ix_bne - ix) * (iy - iy_bne) * gOut;
        }
        if (InBounds3D(iz_tse, iy_tse, ix_tse, in_d, in_h, in_w)) {
          T tse_val = input[inp_offset_NC + iz_tse * inp_sD + iy_tse * inp_sH +
                            ix_tse * inp_sW];
          gix += tse_val * (iy - iy_bnw) * (iz_bnw - iz) * gOut;
          giy += tse_val * (ix - ix_bnw) * (iz_bnw - iz) * gOut;
          giz -= tse_val * (ix - ix_bnw) * (iy - iy_bnw) * gOut;
        }
        if (InBounds3D(iz_bnw, iy_bnw, ix_bnw, in_d, in_h, in_w)) {
          T bnw_val = input[inp_offset_NC + iz_bnw * inp_sD + iy_bnw * inp_sH +
                            ix_bnw * inp_sW];
          gix -= bnw_val * (iy_tse - iy) * (iz - iz_tse) * gOut;
          giy -= bnw_val * (ix_tse - ix) * (iz - iz_tse) * gOut;
          giz += bnw_val * (ix_tse - ix) * (iy_tse - iy) * gOut;
        }
        if (InBounds3D(iz_bne, iy_bne, ix_bne, in_d, in_h, in_w)) {
          T bne_val = input[inp_offset_NC + iz_bne * inp_sD + iy_bne * inp_sH +
                            ix_bne * inp_sW];
          gix += bne_val * (iy_tsw - iy) * (iz - iz_tsw) * gOut;
          giy -= bne_val * (ix - ix_tsw) * (iz - iz_tsw) * gOut;
          giz += bne_val * (ix - ix_tsw) * (iy_tsw - iy) * gOut;
        }
        if (InBounds3D(iz_bsw, iy_bsw, ix_bsw, in_d, in_h, in_w)) {
          T bsw_val = input[inp_offset_NC + iz_bsw * inp_sD + iy_bsw * inp_sH +
                            ix_bsw * inp_sW];
          gix -= bsw_val * (iy - iy_tne) * (iz - iz_tne) * gOut;
          giy += bsw_val * (ix_tne - ix) * (iz - iz_tne) * gOut;
          giz += bsw_val * (ix_tne - ix) * (iy - iy_tne) * gOut;
        }
        if (InBounds3D(iz_bse, iy_bse, ix_bse, in_d, in_h, in_w)) {
          T bse_val = input[inp_offset_NC + iz_bse * inp_sD + iy_bse * inp_sH +
                            ix_bse * inp_sW];
          gix += bse_val * (iy - iy_tnw) * (iz - iz_tnw) * gOut;
          giy += bse_val * (ix - ix_tnw) * (iz - iz_tnw) * gOut;
          giz += bse_val * (ix - ix_tnw) * (iy - iy_tnw) * gOut;
        }
      }
      if (grad_grid != nullptr) {
        T* gGrid_ptr_NDHW = grad_grid + index * grid_sW;
        gGrid_ptr_NDHW[0] = gix_mult * gix;
        gGrid_ptr_NDHW[1] = giy_mult * giy;
        gGrid_ptr_NDHW[2] = giz_mult * giz;
      }
    } else if (mode == Mode::nearest) {
      auto ix_nearest = static_cast<int>(std::round(ix));
      auto iy_nearest = static_cast<int>(std::round(iy));
      auto iz_nearest = static_cast<int>(std::round(iz));

      // assign nearest neighor pixel value to output pixel
      int gOut_offset = n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      for (int c = 0; c < out_c;
           ++c, gOut_offset += gOut_sC, gInp_ptr_NC += inp_sC) {
        AtomicAdd3D(gInp_ptr_NC,
                    iz_nearest,
                    iy_nearest,
                    ix_nearest,
                    inp_sD,
                    inp_sH,
                    inp_sW,
                    in_d,
                    in_h,
                    in_w,
                    grad_output[gOut_offset]);
      }
      if (grad_grid != nullptr) {
        T* gGrid_ptr_NDHW = grad_grid + index * grid_sW;
        gGrid_ptr_NDHW[0] = static_cast<T>(0);
        gGrid_ptr_NDHW[1] = static_cast<T>(0);
        gGrid_ptr_NDHW[2] = static_cast<T>(0);
      }
    }
  }
}

template <typename T, typename Context>
void GridSampleGradKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& grid,
                          const DenseTensor& out_grad,
                          const std::string& mode,
                          const std::string& padding_mode,
                          bool align_corners,
                          DenseTensor* x_grad,
                          DenseTensor* grid_grad) {
  PaddingMode enum_padding_mode;
  Mode enum_mode;
  if (padding_mode == "border") {
    enum_padding_mode = PaddingMode::border;
  } else if (padding_mode == "reflection") {
    enum_padding_mode = PaddingMode::reflect;
  } else {
    enum_padding_mode = PaddingMode::zeros;
  }

  if (mode == "nearest") {
    enum_mode = Mode::nearest;
  } else {
    enum_mode = Mode::bilinear;
  }

  if (x.dims().size() == 4) {
    const int n = grid.dims()[0];
    const int out_h = grid.dims()[1];
    const int out_w = grid.dims()[2];
    const int c = x.dims()[1];
    const int in_h = x.dims()[2];
    const int in_w = x.dims()[3];

    dev_ctx.template Alloc<T>(x_grad);
    phi::funcs::SetConstant<Context, T>()(dev_ctx, x_grad, static_cast<T>(0));

    T* grid_grad_data = nullptr;
    if (grid_grad != nullptr) {
      grid_grad_data = dev_ctx.template Alloc<T>(grid_grad);
    }

    int count = static_cast<int>(n * out_h * out_w);
    auto cu_stream = dev_ctx.stream();
    backends::gpu::GpuLaunchConfig config =
        backends::gpu::GetGpuLaunchConfig1D(dev_ctx, count);
    GridSamplerCudaBackwardKernel<T>
        <<<config.block_per_grid, config.thread_per_block, 0, cu_stream>>>(
            count,
            out_grad.data<T>(),
            x.data<T>(),
            grid.data<T>(),
            n,
            c,
            out_h,
            out_w,
            in_h,
            in_w,
            x_grad->data<T>(),
            grid_grad_data,
            enum_mode,
            enum_padding_mode,
            align_corners);
  } else {
    const int out_d = grid.dims()[1];
    const int out_h = grid.dims()[2];
    const int out_w = grid.dims()[3];
    const int n = x.dims()[0];
    const int c = x.dims()[1];
    const int in_d = x.dims()[2];
    const int in_h = x.dims()[3];
    const int in_w = x.dims()[4];

    dev_ctx.template Alloc<T>(x_grad);
    phi::funcs::SetConstant<Context, T>()(dev_ctx, x_grad, static_cast<T>(0));

    T* grid_grad_data = nullptr;
    if (grid_grad != nullptr) {
      grid_grad_data = dev_ctx.template Alloc<T>(grid_grad);
    }

    int count = static_cast<int>(n * out_d * out_h * out_w);
    auto cu_stream = dev_ctx.stream();
    backends::gpu::GpuLaunchConfig config =
        backends::gpu::GetGpuLaunchConfig1D(dev_ctx, count);
    GridSampler3DCudaBackwardKernel<T>
        <<<config.block_per_grid, config.thread_per_block, 0, cu_stream>>>(
            count,
            out_grad.data<T>(),
            x.data<T>(),
            grid.data<T>(),
            c,
            out_d,
            out_h,
            out_w,
            in_d,
            in_h,
            in_w,
            x_grad->data<T>(),
            grid_grad_data,
            enum_mode,
            enum_padding_mode,
            align_corners);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(grid_sample_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::GridSampleGradKernel,
                   float,
                   double) {}
