#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/grid_sample_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/grid_sample_utils.h"

namespace phi {

template <typename T>
static __forceinline__ __device__ void AtomicAdd(
    T* data, int h, int w, int sH, int sW, int H, int W, T delta) {
  if (InBounds(h, w, H, W)) {
    paddle::platform::CudaAtomicAdd(data + h * sH + w * sW, delta);
  }
}

template <typename T>
static __forceinline__ __device__ T
UnnormalizeWithMask(T coord, int size, bool align_corners, T* grad_in) {
  if (align_corners) {
    *grad_in = static_cast<T>(size - 1) / 2;
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    *grad_in = static_cast<T>(size) / 2;
    return ((coord + 1.f) * size - 1) / 2;
  }
}

template <typename T>
static __forceinline__ __device__ T ClipIndexesWithMask(T in,
                                                        int clip_limit,
                                                        T* grad_in) {
  if (in <= static_cast<T>(0)) {
    *grad_in = static_cast<T>(0);
    return static_cast<T>(0);
  } else {
    T max = static_cast<T>(clip_limit - 1);
    if (in >= max) {
      *grad_in = static_cast<T>(0);
      return max;
    } else {
      *grad_in = static_cast<T>(1);
      return in;
    }
  }
}

template <typename T>
static __forceinline__ __device__ T
ReflectIndexesWithMask(T in, int twice_low, int twice_high, T* grad_in) {
  if (twice_low == twice_high) {
    *grad_in = static_cast<T>(0);
    return static_cast<T>(0);
  }
  int grad_in_mult_;
  T min = static_cast<T>(twice_low) / 2;
  T span = static_cast<T>(twice_high - twice_low) / 2;
  in = in - min;
  if (in < static_cast<T>(0)) {
    grad_in_mult_ = -1;
    in = -in;
  } else {
    grad_in_mult_ = 1;
  }
  T extra = fmod(in, span);
  int flips = static_cast<int>(floor(in / span));
  if (flips % 2 == 0) {
    *grad_in = static_cast<T>(grad_in_mult_);
    return extra + min;
  } else {
    *grad_in = static_cast<T>(-grad_in_mult_);
    return span - extra + min;
  }
}

template <typename T>
static __forceinline__ __device__ T
ComputePositionsWithMask(T coord,
                         int size,
                         PaddingMode padding_mode,
                         bool align_corners,
                         T* grad_in) {
  T grad_clip, grad_refl;
  coord = UnnormalizeWithMask<T>(coord, size, align_corners, grad_in);
  if (padding_mode == PaddingMode::border) {
    coord = ClipIndexesWithMask(coord, size, &grad_clip);
    *grad_in = (*grad_in) * grad_clip;
  } else if (padding_mode == PaddingMode::reflect) {
    if (align_corners) {
      coord = ReflectIndexesWithMask(coord, 0, 2 * (size - 1), &grad_refl);
    } else {
      coord = ReflectIndexesWithMask(coord, -1, 2 * size - 1, &grad_refl);
    }
    coord = ClipIndexesWithMask(coord, size, &grad_clip);
    *grad_in = (*grad_in) * grad_refl * grad_clip;
  }

  return coord;
}

template <typename T>
__global__ void GridSamplerCudaBackwardKernel(const int nthreads,
                                              const T* grad_output,
                                              const T* input,
                                              const T* grid,
                                              int n,
                                              int out_c,
                                              int out_h,
                                              int out_w,
                                              int in_h,
                                              int in_w,
                                              T* grad_input,
                                              T* grad_grid,
                                              const Mode mode,
                                              const PaddingMode padding_mode,
                                              bool align_corners) {
  int inp_sN = out_c * in_h * in_w;
  int inp_sC = in_h * in_w;
  int inp_sH = in_w;
  int inp_sW = 1;
  int grid_sN = out_h * out_w * 2;
  int grid_sH = out_w * 2;
  int grid_sW = 2;
  int grid_sCoor = 1;

  int gOut_sN = out_c * out_h * out_w;
  int gOut_sC = out_h * out_w;
  int gOut_sH = out_w;
  int gOut_sW = 1;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_w;
    const int h = (index / out_w) % out_h;
    const int n = index / (out_h * out_w);
    const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];

    T gix_mult, giy_mult;
    ix = ComputePositionsWithMask(
        ix, in_w, padding_mode, align_corners, &gix_mult);
    iy = ComputePositionsWithMask(
        iy, in_h, padding_mode, align_corners, &giy_mult);

    if (mode == Mode::bilinear) {
      int ix_nw = static_cast<int>(floor(ix));
      int iy_nw = static_cast<int>(floor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      T nw = (ix_se - ix) * (iy_se - iy);
      T ne = (ix - ix_sw) * (iy_sw - iy);
      T sw = (ix_ne - ix) * (iy - iy_ne);
      T se = (ix - ix_nw) * (iy - iy_nw);

      T gix = static_cast<T>(0), giy = static_cast<T>(0);
      int gOut_offset = n * gOut_sN + h * gOut_sH + w * gOut_sW;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      int inp_offset_NC = n * inp_sN;
      for (int c = 0; c < out_c; ++c,
               inp_offset_NC += inp_sC,
               gInp_ptr_NC += inp_sC,
               gOut_offset += gOut_sC) {
        T gOut = grad_output[gOut_offset];

        AtomicAdd(
            gInp_ptr_NC, iy_nw, ix_nw, inp_sH, inp_sW, in_h, in_w, nw * gOut);
        AtomicAdd(
            gInp_ptr_NC, iy_ne, ix_ne, inp_sH, inp_sW, in_h, in_w, ne * gOut);
        AtomicAdd(
            gInp_ptr_NC, iy_sw, ix_sw, inp_sH, inp_sW, in_h, in_w, sw * gOut);
        AtomicAdd(
            gInp_ptr_NC, iy_se, ix_se, inp_sH, inp_sW, in_h, in_w, se * gOut);

        if (InBounds(iy_nw, ix_nw, in_h, in_w)) {
          T nw_val = input[inp_offset_NC + iy_nw * inp_sH + ix_nw * inp_sW];
          gix -= nw_val * (iy_se - iy) * gOut;
          giy -= nw_val * (ix_se - ix) * gOut;
        }
        if (InBounds(iy_ne, ix_ne, in_h, in_w)) {
          T ne_val = input[inp_offset_NC + iy_ne * inp_sH + ix_ne * inp_sW];
          gix += ne_val * (iy_sw - iy) * gOut;
          giy -= ne_val * (ix - ix_sw) * gOut;
        }
        if (InBounds(iy_sw, ix_sw, in_h, in_w)) {
          T sw_val = input[inp_offset_NC + iy_sw * inp_sH + ix_sw * inp_sW];
          gix -= sw_val * (iy - iy_ne) * gOut;
          giy += sw_val * (ix_ne - ix) * gOut;
        }
        if (InBounds(iy_se, ix_se, in_h, in_w)) {
          T se_val = input[inp_offset_NC + iy_se * inp_sH + ix_se * inp_sW];
          gix += se_val * (iy - iy_nw) * gOut;
          giy += se_val * (ix - ix_nw) * gOut;
        }
      }

      if (grad_grid != nullptr) {
        T* gGrid_ptr_NHW = grad_grid + index * grid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      }
    } else if (mode == Mode::nearest) {
      int ix_nearest = static_cast<int>(std::nearbyint(ix));
      int iy_nearest = static_cast<int>(std::nearbyint(iy));

      int gOut_offset = n * gOut_sN + h * gOut_sH + w * gOut_sW;
      T* gInp_ptr_NC = grad_input + n * inp_sN;
      for (int c = 0; c < out_c;
           ++c, gInp_ptr_NC += inp_sC, gOut_offset += gOut_sC) {
        AtomicAdd(gInp_ptr_NC,
                  iy_nearest,
                  ix_nearest,
                  inp_sH,
                  inp_sW,
                  in_h,
                  in_w,
                  grad_output[gOut_offset]);
      }

      if (grad_grid != nullptr) {
        T* gGrid_ptr_NHW = grad_grid + index * grid_sW;
        gGrid_ptr_NHW[0] = static_cast<T>(0);
        gGrid_ptr_NHW[1] = static_cast<T>(0);
      }
    }
  }
}

template <typename T, typename Context>
void GridSampleGradKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& grid,
                          const DenseTensor& out_grad,
                          const std::string& mode,
                          const std::string& padding_mode,
                          bool align_corners,
                          DenseTensor* x_grad,
                          DenseTensor* grid_grad) {
  PaddingMode enum_padding_mode;
  Mode enum_mode;
  if (padding_mode == "border") {
    enum_padding_mode = PaddingMode::border;
  } else if (padding_mode == "reflection") {
    enum_padding_mode = PaddingMode::reflect;
  } else {
    enum_padding_mode = PaddingMode::zeros;
  }

  if (mode == "nearest") {
    enum_mode = Mode::nearest;
  } else {
    enum_mode = Mode::bilinear;
  }

  const int n = grid.dims()[0];
  const int out_h = grid.dims()[1];
  const int out_w = grid.dims()[2];
  const int c = x.dims()[1];
  const int in_h = x.dims()[2];
  const int in_w = x.dims()[3];

  dev_ctx.template Alloc<T>(x_grad);
  phi::funcs::SetConstant<Context, T>()(dev_ctx, x_grad, static_cast<T>(0));

  T* grid_grad_data = nullptr;
  if (grid_grad != nullptr) {
    grid_grad_data = dev_ctx.template Alloc<T>(grid_grad);
  }

  int count = static_cast<int>(n * out_h * out_w);
  auto cu_stream = dev_ctx.stream();
  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, count);
  GridSamplerCudaBackwardKernel<T>
      <<<config.block_per_grid, config.thread_per_block, 0, cu_stream>>>(
          count,
          out_grad.data<T>(),
          x.data<T>(),
          grid.data<T>(),
          n,
          c,
          out_h,
          out_w,
          in_h,
          in_w,
          x_grad->data<T>(),
          grid_grad_data,
          enum_mode,
          enum_padding_mode,
          align_corners);
}

}  // namespace phi

PD_REGISTER_KERNEL(grid_sample_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::GridSampleGradKernel,
                   float,
                   double) {}
