#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/uniform_random_kernel.h"

#include "gflags/gflags.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/distribution_helper.h"
#include "paddle/phi/kernels/funcs/index_impl.cu.h"

DECLARE_bool(use_curand);

namespace phi {

template <typename T>
struct UniformGenerator {
  T min_, max_;
  unsigned int seed_;
  T diag_val_;
  unsigned int diag_num_;
  unsigned int diag_step_;
  __host__ __device__ UniformGenerator(
      T min, T max, int seed, int diag_num, int diag_step, T diag_val)
      : min_(min),
        max_(max),
        seed_(seed),
        diag_num_(diag_num),
        diag_step_(diag_step),
        diag_val_(diag_val) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n);
    T out = dist(rng);
    unsigned int remainder = n % (diag_step_ + 1);
    if (remainder == 0 && diag_num_ > n / (diag_step_ + 1)) {
      out = diag_val_;
    }
    return out;
  }
};

template <typename T>
struct UniformGeneratorOffset {
  T min_, max_;
  unsigned int seed_;
  T diag_val_;
  unsigned int diag_num_;
  unsigned int diag_step_;
  int offset_;
  __host__ __device__ UniformGeneratorOffset(T min,
                                             T max,
                                             int seed,
                                             int diag_num,
                                             int diag_step,
                                             T diag_val,
                                             int offset)
      : min_(min),
        max_(max),
        seed_(seed),
        diag_num_(diag_num),
        diag_step_(diag_step),
        diag_val_(diag_val),
        offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n + offset_);
    T out = dist(rng);
    unsigned int remainder = n % (diag_step_ + 1);
    if (remainder == 0 && diag_num_ > n / (diag_step_ + 1)) {
      out = diag_val_;
    }
    return out;
  }
};

template <typename T, typename Context>
void UniformRandomRawKernel(const Context& dev_ctx,
                            const ScalarArray& shape,
                            DataType dtype,
                            float min,
                            float max,
                            int seed,
                            int diag_num,
                            int diag_step,
                            float diag_val,
                            DenseTensor* out) {
  out->Resize(phi::make_ddim(shape.GetData()));
  T* data = dev_ctx.template Alloc<T>(out);
  auto size = out->numel();
  bool seed_flag = false;
  if (seed == 0) {
    std::random_device rd;
    seed = rd();
    seed_flag = true;
  }

  auto generator = dev_ctx.GetGenerator();
  if (generator->GetIsInitPy() && seed_flag) {
    if (FLAGS_use_curand) {
      using MT = typename kps::details::MPTypeTrait<T>::Type;
      funcs::uniform_distribution<MT> dist;
      funcs::uniform_real_transform<MT> trans(min, max);
      funcs::distribution_and_transform<T>(dev_ctx, out, dist, trans);
    } else {
      auto seed_offset = generator->IncrementOffset(1);
      int64_t gen_offset = size * seed_offset.second;
      auto func = UniformGeneratorOffset<T>(min,
                                            max,
                                            seed_offset.first,
                                            diag_num,
                                            diag_step,
                                            diag_val,
                                            gen_offset);
      IndexKernel<T, UniformGeneratorOffset<T>>(dev_ctx, out, func);
    }
  } else {
    auto func =
        UniformGenerator<T>(min, max, seed, diag_num, diag_step, diag_val);
    IndexKernel<T, UniformGenerator<T>>(dev_ctx, out, func);
  }
}

template <typename T, typename Context>
void UniformRandomKernel(const Context& dev_ctx,
                         const ScalarArray& shape,
                         DataType dtype,
                         float min,
                         float max,
                         int seed,
                         DenseTensor* out) {
  UniformRandomRawKernel<T>(
      dev_ctx, shape, dtype, min, max, seed, 0, 0, 0.0f, out);
}

}  // namespace phi

PD_REGISTER_KERNEL(uniform_random_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::UniformRandomRawKernel,
                   float,
                   double) {}

PD_REGISTER_KERNEL(
    uniform_random, GPU, ALL_LAYOUT, phi::UniformRandomKernel, float, double) {}
