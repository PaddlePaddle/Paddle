#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/adamw_kernel.h"

#include <math.h>  // for sqrt in CPU and CUDA

#include <vector>

#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/adam_functors.h"
#include "paddle/phi/kernels/funcs/for_range.h"

namespace phi {
template <typename T, typename MT>
__global__ void AdamWKernelREG(MT beta1,
                               MT beta2,
                               MT epsilon,
                               MT coeff,
                               MT lr_ratio,
                               MT beta1_pow_,
                               MT beta2_pow_,
                               const MT* moment1,
                               MT* moment1_out,
                               const MT* moment2,
                               MT* moment2_out,
                               const MT* lr_,
                               const T* grad,
                               const T* param,
                               T* param_out,
                               const MT* master_param,
                               MT* master_param_out,
                               int ndim) {
  MT lr = *lr_ * lr_ratio;
  MT beta1_pow = beta1_pow_;
  MT beta2_pow = beta2_pow_;

  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (; id < ndim; id += gridDim.x * blockDim.x) {
    MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
    MT g = static_cast<MT>(grad[id]);
    MT mom1 = static_cast<MT>(moment1[id]);
    MT mom2 = static_cast<MT>(moment2[id]);

    p *= (static_cast<MT>(1.0) - lr * coeff);

    mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
    mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g * g;

    MT denom = (sqrt(mom2) / sqrt(static_cast<MT>(1.0) - beta2_pow)) + epsilon;

    p += (mom1 / denom) * (-(lr / (static_cast<MT>(1.0) - beta1_pow)));

    moment1_out[id] = mom1;
    moment2_out[id] = mom2;
    param_out[id] = static_cast<T>(p);
    if (master_param_out) {
      master_param_out[id] = p;
    }
  }
}

template <typename T, typename MT>
__global__ void AdamWKernelMEM(MT beta1,
                               MT beta2,
                               MT epsilon,
                               MT coeff,
                               MT lr_ratio,
                               const MT* beta1_pow_,
                               const MT* beta2_pow_,
                               const MT* moment1,
                               MT* moment1_out,
                               const MT* moment2,
                               MT* moment2_out,
                               const MT* lr_,
                               const T* grad,
                               const T* param,
                               T* param_out,
                               const MT* master_param,
                               MT* master_param_out,
                               int ndim) {
  MT lr = *lr_ * lr_ratio;
  MT beta1_pow = *beta1_pow_;
  MT beta2_pow = *beta2_pow_;

  int id = blockIdx.x * blockDim.x + threadIdx.x;

  for (; id < ndim; id += gridDim.x * blockDim.x) {
    MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
    MT g = static_cast<MT>(grad[id]);
    MT mom1 = static_cast<MT>(moment1[id]);
    MT mom2 = static_cast<MT>(moment2[id]);

    p *= (static_cast<MT>(1.0) - lr * coeff);

    mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
    mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g * g;

    MT denom = (sqrt(mom2) / sqrt(static_cast<MT>(1.0) - beta2_pow)) + epsilon;

    p += (mom1 / denom) * (-(lr / (static_cast<MT>(1.0) - beta1_pow)));

    moment1_out[id] = mom1;
    moment2_out[id] = mom2;
    param_out[id] = static_cast<T>(p);
    if (master_param_out) {
      master_param_out[id] = p;
    }
  }
}

template <typename T>
__global__ void UpdateAdamWBetaPow(T beta1,
                                   T beta2,
                                   const T* beta1_pow_,
                                   const T* beta2_pow_,
                                   T* beta1_pow_out,
                                   T* beta2_pow_out) {
  *beta1_pow_out = beta1 * beta1_pow_[0];
  *beta2_pow_out = beta2 * beta2_pow_[0];
}

template <typename T, typename Context>
void AdamwDenseKernel(const Context& dev_ctx,
                      const DenseTensor& param,
                      const DenseTensor& grad,
                      const DenseTensor& learning_rate,
                      const DenseTensor& moment1,
                      const DenseTensor& moment2,
                      const DenseTensor& beta1_pow,
                      const DenseTensor& beta2_pow,
                      const paddle::optional<DenseTensor>& master_param,
                      const paddle::optional<DenseTensor>& skip_update,
                      const Scalar& beta1,
                      const Scalar& beta2,
                      const Scalar& epsilon,
                      float lr_ratio,
                      float coeff,
                      bool with_decay,
                      bool lazy_mode,
                      int64_t min_row_size_to_use_multithread,
                      bool multi_precision,
                      bool use_global_beta_pow,
                      DenseTensor* param_out,
                      DenseTensor* moment1_out,
                      DenseTensor* moment2_out,
                      DenseTensor* beta1_pow_out,
                      DenseTensor* beta2_pow_out,
                      DenseTensor* master_param_outs) {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;

  VLOG(4) << "use_global_beta_pow:" << use_global_beta_pow;

  MPDType coeff_ = static_cast<MPDType>(coeff);
  MPDType lr_ratio_ = static_cast<MPDType>(lr_ratio);

  bool skip_update_ = false;
  if (skip_update.is_initialized()) {
    PADDLE_ENFORCE_EQ(
        skip_update->numel(),
        1,
        errors::InvalidArgument("Input(SkipUpdate) size must be 1, but get %d",
                                skip_update->numel()));
    std::vector<bool> skip_update_vec;
    paddle::framework::TensorToVector(*skip_update, dev_ctx, &skip_update_vec);
    skip_update_ = skip_update_vec[0];
  }

  // skip_update=true, just copy input to output, and TensorCopy will call
  // mutable_data
  if (skip_update_) {
    VLOG(4) << "Adamw skip update";
    phi::Copy(dev_ctx, param, dev_ctx.GetPlace(), false, param_out);
    phi::Copy(dev_ctx, moment1, dev_ctx.GetPlace(), false, moment1_out);
    phi::Copy(dev_ctx, moment2, dev_ctx.GetPlace(), false, moment2_out);
    phi::Copy(dev_ctx, beta1_pow, beta1_pow.place(), false, beta1_pow_out);
    phi::Copy(dev_ctx, beta2_pow, beta2_pow.place(), false, beta2_pow_out);
    return;
  }

  // if with_decay = false, coeff = 0
  if (!with_decay) {
    coeff_ = static_cast<MPDType>(0.0);
  }

  MPDType beta1_ = beta1.to<MPDType>();
  MPDType beta2_ = beta2.to<MPDType>();
  MPDType epsilon_ = epsilon.to<MPDType>();
  VLOG(3) << "beta1_pow.numel() : " << beta1_pow.numel()
          << "beta2_pow.numel() : " << beta2_pow.numel();
  VLOG(3) << "param.numel(): " << param.numel();
  PADDLE_ENFORCE_EQ(
      beta1_pow_out->numel(),
      1,
      errors::InvalidArgument("beta1 pow output size should be 1, but received "
                              "value is:%d.",
                              beta1_pow_out->numel()));

  PADDLE_ENFORCE_EQ(
      beta2_pow_out->numel(),
      1,
      errors::InvalidArgument("beta2 pow output size should be 1, but received "
                              "value is:%d.",
                              beta2_pow_out->numel()));

  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision ? dev_ctx.template Alloc<MPDType>(master_param_outs)
                      : nullptr;

  // update param and moment
  int threads = 512;
  int blocks = (param.numel() + threads - 1) / threads;

  if (beta1_pow.place() == CPUPlace() && beta2_pow.place() == CPUPlace()) {
    // Compute with betapow in REG
    AdamWKernelREG<T, MPDType><<<blocks, threads, 0, dev_ctx.stream()>>>(
        beta1_,
        beta2_,
        epsilon_,
        coeff_,
        lr_ratio_,
        *beta1_pow.data<MPDType>(),
        *beta2_pow.data<MPDType>(),
        moment1.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(moment1_out),
        moment2.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(moment2_out),
        learning_rate.data<MPDType>(),
        grad.data<T>(),
        param.data<T>(),
        dev_ctx.template Alloc<T>(param_out),
        master_in_data,
        master_out_data,
        param.numel());
    if (!use_global_beta_pow) {
      // Cpu update
      dev_ctx.template HostAlloc<MPDType>(beta1_pow_out)[0] =
          beta1_ * beta1_pow.data<MPDType>()[0];
      dev_ctx.template HostAlloc<MPDType>(beta2_pow_out)[0] =
          beta2_ * beta2_pow.data<MPDType>()[0];
    }
  } else {
    AdamWKernelMEM<T, MPDType><<<blocks, threads, 0, dev_ctx.stream()>>>(
        beta1_,
        beta2_,
        epsilon_,
        coeff_,
        lr_ratio_,
        beta1_pow.data<MPDType>(),
        beta2_pow.data<MPDType>(),
        moment1.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(moment1_out),
        moment2.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(moment2_out),
        learning_rate.data<MPDType>(),
        grad.data<T>(),
        param.data<T>(),
        dev_ctx.template Alloc<T>(param_out),
        master_in_data,
        master_out_data,
        param.numel());
    if (!use_global_beta_pow) {
      // Update with gpu
      UpdateAdamWBetaPow<MPDType><<<1, 32, 0, dev_ctx.stream()>>>(
          beta1_,
          beta2_,
          beta1_pow.data<MPDType>(),
          beta2_pow.data<MPDType>(),
          dev_ctx.template Alloc<MPDType>(beta1_pow_out),
          dev_ctx.template Alloc<MPDType>(beta2_pow_out));
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(adamw,
                   GPU,
                   ALL_LAYOUT,
                   phi::AdamwDenseKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  // Skip beta1_pow, beta2_pow, skip_update data transform
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(6).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(8).SetBackend(phi::Backend::ALL_BACKEND);
}
