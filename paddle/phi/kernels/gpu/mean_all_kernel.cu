// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/mean_all_kernel.h"

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

namespace phi {

template <typename T, typename Context>
void MeanAllKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   DenseTensor* out) {
  const T* in_data = x.data<T>();
  T* out_data = dev_ctx.template Alloc<T>(out);
  auto numel = x.numel();
  auto rank = x.dims().size();
  auto place = dev_ctx.GetPlace();
  auto stream = dev_ctx.stream();

  if (rank == 0) {  // scalar
    paddle::memory::Copy(
        place, out_data, place, in_data, numel * sizeof(T), stream);
    return;
  }

  std::vector<int> reduce_dims;
  reduce_dims.reserve(rank);
  for (decltype(rank) i = 0; i < rank; ++i) {
    reduce_dims.push_back(i);
  }
  funcs::ReduceKernel<T, T, kps::AddFunctor, kps::IdentityFunctor<T>>(
      dev_ctx,
      x,
      out,
      kps::IdentityFunctor<T>(),
      reduce_dims,
      /*is_mean=*/true);
}

}  // namespace phi

PD_REGISTER_KERNEL(mean_all,
                   GPU,
                   ALL_LAYOUT,
                   phi::MeanAllKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
