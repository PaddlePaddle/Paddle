/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/c_identity_kernel.h"
#include "paddle/phi/kernels/impl/c_identity_kernel_impl.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

#if (NCCL_VERSION_CODE >= 21000 && CUDA_VERSION >= 11000) || defined(PADDLE_WITH_HIP)
PD_REGISTER_KERNEL(c_identity,
                   GPU,
                   ALL_LAYOUT,
                   phi::CIdentityKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(c_identity,
                   GPU,
                   ALL_LAYOUT,
                   phi::CIdentityKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
#endif
