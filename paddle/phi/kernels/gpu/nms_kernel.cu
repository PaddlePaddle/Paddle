#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/nms_kernel.h"

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

static const int64_t threadsPerBlock = sizeof(int64_t) * 8;

namespace phi {

template <typename T>
static __global__ void NMS(const T* boxes_data,
                           float threshold,
                           int64_t num_boxes,
                           uint64_t* masks) {
  auto raw_start = blockIdx.y;
  auto col_start = blockIdx.x;
  if (raw_start > col_start) return;

  const int raw_last_storage =
      min(num_boxes - raw_start * threadsPerBlock, threadsPerBlock);
  const int col_last_storage =
      min(num_boxes - col_start * threadsPerBlock, threadsPerBlock);

  if (threadIdx.x < raw_last_storage) {
    uint64_t mask = 0;
    auto current_box_idx = raw_start * threadsPerBlock + threadIdx.x;
    const T* current_box = boxes_data + current_box_idx * 4;
    for (int i = 0; i < col_last_storage; ++i) {
      const T* target_box = boxes_data + (col_start * threadsPerBlock + i) * 4;
      if (CalculateIoU<T>(current_box, target_box, threshold)) {
        mask |= 1ULL << i;
      }
    }
    const int blocks_per_line = CeilDivide(num_boxes, threadsPerBlock);
    masks[current_box_idx * blocks_per_line + col_start] = mask;
  }
}

template <typename T, typename Context>
void NMSKernel(const Context& dev_ctx,
               const DenseTensor& boxes,
               float threshold,
               DenseTensor* output) {
  const int64_t num_boxes = boxes.dims()[0];
  const auto blocks_per_line = CeilDivide(num_boxes, threadsPerBlock);
  dim3 block(threadsPerBlock);
  dim3 grid(blocks_per_line, blocks_per_line);
  auto mask_data = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      num_boxes * blocks_per_line * sizeof(uint64_t),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  uint64_t* mask_dev = reinterpret_cast<uint64_t*>(mask_data->ptr());
  NMS<T><<<grid, block, 0, dev_ctx.stream()>>>(
      boxes.data<T>(), threshold, num_boxes, mask_dev);
  std::vector<uint64_t> mask_host(num_boxes * blocks_per_line);
  paddle::memory::Copy(phi::CPUPlace(),
                       mask_host.data(),
                       dev_ctx.GetPlace(),
                       mask_dev,
                       num_boxes * blocks_per_line * sizeof(uint64_t),
                       dev_ctx.stream());
  std::vector<int64_t> remv(blocks_per_line);
  std::vector<int64_t> keep_boxes_idxs(num_boxes);
  int64_t* output_host = keep_boxes_idxs.data();
  int64_t last_box_num = 0;
  for (int64_t i = 0; i < num_boxes; ++i) {
    auto remv_element_id = i / threadsPerBlock;
    auto remv_bit_id = i % threadsPerBlock;
    if (!(remv[remv_element_id] & 1ULL << remv_bit_id)) {
      output_host[last_box_num++] = i;
      uint64_t* current_mask = mask_host.data() + i * blocks_per_line;
      for (auto j = remv_element_id; j < blocks_per_line; ++j) {
        remv[j] |= current_mask[j];
      }
    }
  }
  output->Resize(phi::make_ddim({last_box_num}));
  auto* output_data = dev_ctx.template Alloc<int64_t>(output);
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       output_data,
                       phi::CPUPlace(),
                       output_host,
                       sizeof(int64_t) * last_box_num,
                       dev_ctx.stream());
}
}  // namespace phi
PD_REGISTER_KERNEL(nms, GPU, ALL_LAYOUT, phi::NMSKernel, float, double) {}
