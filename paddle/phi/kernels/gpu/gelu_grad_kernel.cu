#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/gelu_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/gpu/gelu_funcs.h"

DECLARE_bool(use_fast_math);

namespace phi {

template <typename T>
struct GeluWithApproximateGradFunctor {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x, T arg_dout) {
    MPType x = static_cast<MPType>(arg_x);
    MPType dout = static_cast<MPType>(arg_dout);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    MPType kBeta =
        kAlpha * static_cast<MPType>(GELU_CONSTANT) * static_cast<MPType>(3);
    auto cube_x = x * x * x;
    auto tanh_out =
        tanh(kAlpha * ((static_cast<MPType>(GELU_CONSTANT) * cube_x) + x));
    auto ans =
        half * (one + tanh_out +
                (one - tanh_out * tanh_out) * (x * kAlpha + kBeta * cube_x));
    return static_cast<T>(ans * dout);
  }
};

template <typename T>
struct GeluWithoutApproximateGradFunctor {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x, T arg_dout) {
    MPType x = static_cast<MPType>(arg_x);
    MPType dout = static_cast<MPType>(arg_dout);
    constexpr MPType kBeta = M_2_SQRTPI * M_SQRT1_2 * static_cast<MPType>(0.5);
    const MPType cdf = normcdf(x);
    const MPType pdf = exp(static_cast<MPType>(-0.5) * x * x) * kBeta;
    return static_cast<T>(dout * (cdf + x * pdf));
  }
};

template <typename T, typename Context>
void GeluGradKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& out_grad,
                    bool approximate,
                    DenseTensor* x_grad) {
  dev_ctx.template Alloc<T>(x_grad);
  std::vector<const DenseTensor*> ins = {&x, &out_grad};
  std::vector<DenseTensor*> outs = {x_grad};
  if (approximate) {
#ifdef __NVCC__
    if (std::is_same<T, dtype::float16>::value) {
      size_t n = x.numel();
      const auto* x_ptr = reinterpret_cast<const __half*>(x.data<T>());
      const auto* y_g_ptr = reinterpret_cast<const __half*>(out_grad.data<T>());
      auto* x_g_ptr = reinterpret_cast<__half*>(x_grad->data<T>());
      if (TryLaunchFP16FastGeluBwdVectorizeCUDAKernel(
              dev_ctx, x_ptr, y_g_ptr, x_g_ptr, n)) {
        return;
      }
    }
#endif
    using Functor = GeluWithApproximateGradFunctor<T>;
    phi::funcs::ElementwiseKernel<T, Functor, 1>(
        dev_ctx, ins, &outs, Functor());
  } else {
    using Functor = GeluWithoutApproximateGradFunctor<T>;
    phi::funcs::ElementwiseKernel<T, Functor, 1>(
        dev_ctx, ins, &outs, Functor());
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(gelu_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::GeluGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
