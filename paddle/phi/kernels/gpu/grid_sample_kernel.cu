#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/grid_sample_kernel.h"

#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/grid_sample_utils.h"

namespace phi {

template <typename T>
static __forceinline__ __device__ T Unnormalize(T coord,
                                                int size,
                                                bool align_corners) {
  if (align_corners) {
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    return ((coord + 1.f) * size - 1) / 2;
  }
}

template <typename T>
static __forceinline__ __device__ T ClipIndexes(T in, int max_value) {
  return min(static_cast<T>(max_value), max(in, static_cast<T>(0)));
}

template <typename T>
static __forceinline__ __device__ T ReflectIndexes(T in,
                                                   int twice_low,
                                                   int twice_high) {
  if (twice_low == twice_high) {
    return static_cast<T>(0);
  }
  T min = static_cast<T>(twice_low) / 2;
  T span = static_cast<T>(twice_high - twice_low) / 2;
  in = fabs(in - min);
  T extra = fmod(in, span);
  int flips = static_cast<int>(floor(in / span));
  if (flips % 2 == 0) {
    return extra + min;
  } else {
    return span - extra + min;
  }
}

template <typename T>
static __forceinline__ __device__ T ComputePositions(T coord,
                                                     int size,
                                                     PaddingMode padding_mode,
                                                     bool align_corners) {
  coord = Unnormalize<T>(coord, size, align_corners);
  if (padding_mode == PaddingMode::border) {
    coord = ClipIndexes(coord, size - 1);
  } else if (padding_mode == PaddingMode::reflect) {
    if (align_corners) {
      coord = ReflectIndexes(coord, 0, 2 * (size - 1));
    } else {
      coord = ReflectIndexes(coord, -1, 2 * size - 1);
    }
    coord = ClipIndexes(coord, size - 1);
  }
  return coord;
}

template <typename T>
__global__ void GridSampleCudaKernel(const int nthreads,
                                     int n,
                                     int out_c,
                                     int out_h,
                                     int out_w,
                                     int in_h,
                                     int in_w,
                                     const T* input,
                                     const T* grid,
                                     T* output,
                                     const Mode mode,
                                     const PaddingMode padding_mode,
                                     bool align_corners) {
  int inp_sN = out_c * in_h * in_w;

  int inp_sC = in_h * in_w;
  int inp_sH = in_w;
  int inp_sW = 1;
  int grid_sN = out_h * out_w * 2;
  int grid_sH = out_w * 2;
  int grid_sW = 2;
  int grid_sCoor = 1;
  int out_sN = out_c * out_h * out_w;
  int out_sC = out_h * out_w;
  int out_sH = out_w;
  int out_sW = 1;
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_w;
    const int h = (index / out_w) % out_h;
    const int n = index / (out_h * out_w);
    const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];

    ix = ComputePositions(ix, in_w, padding_mode, align_corners);
    iy = ComputePositions(iy, in_h, padding_mode, align_corners);
    if (mode == Mode::bilinear) {
      int ix_nw = static_cast<int>(floor(ix));
      int iy_nw = static_cast<int>(floor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      T nw = (ix_se - ix) * (iy_se - iy);
      T ne = (ix - ix_sw) * (iy_sw - iy);
      T sw = (ix_ne - ix) * (iy - iy_ne);
      T se = (ix - ix_nw) * (iy - iy_nw);

      auto inp_offset_NC = n * inp_sN;

      auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < out_c;
           ++c, inp_offset_NC += inp_sC, out_ptr_NCHW += out_sC) {
        *out_ptr_NCHW = static_cast<T>(0);
        if (InBounds(iy_nw, ix_nw, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_nw * inp_sH + ix_nw * inp_sW] * nw;
        }
        if (InBounds(iy_ne, ix_ne, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_ne * inp_sH + ix_ne * inp_sW] * ne;
        }
        if (InBounds(iy_sw, ix_sw, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_sw * inp_sH + ix_sw * inp_sW] * sw;
        }
        if (InBounds(iy_se, ix_se, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_se * inp_sH + ix_se * inp_sW] * se;
        }
      }
    } else if (mode == Mode::nearest) {
      int ix_nearest = static_cast<int>(std::nearbyint(ix));
      int iy_nearest = static_cast<int>(std::nearbyint(iy));
      auto inp_offset_NC = n * inp_sN;
      auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < out_c;
           ++c, inp_offset_NC += inp_sC, out_ptr_NCHW += out_sC) {
        if (InBounds(iy_nearest, ix_nearest, in_h, in_w)) {
          *out_ptr_NCHW =
              input[inp_offset_NC + iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr_NCHW = static_cast<T>(0);
        }
      }
    }
  }
}

template <typename T, typename Context>
void GridSampleKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      const DenseTensor& grid,
                      const std::string& mode,
                      const std::string& padding_mode,
                      bool align_corners,
                      DenseTensor* out) {
  PaddingMode enum_padding_mode;
  Mode enum_mode;
  if (padding_mode == "border") {
    enum_padding_mode = PaddingMode::border;
  } else if (padding_mode == "reflection") {
    enum_padding_mode = PaddingMode::reflect;
  } else {
    enum_padding_mode = PaddingMode::zeros;
  }

  if (mode == "nearest") {
    enum_mode = Mode::nearest;
  } else {
    enum_mode = Mode::bilinear;
  }

  const int n = grid.dims()[0];
  const int out_h = grid.dims()[1];
  const int out_w = grid.dims()[2];
  const int c = x.dims()[1];
  const int in_h = x.dims()[2];
  const int in_w = x.dims()[3];
  VLOG(3) << "n: " << n << "; c: " << c << "; out_h: " << out_h
          << "; out_w: " << out_w;

  auto* output_data = dev_ctx.template Alloc<T>(out);
  VLOG(3) << "out dims: " << out->dims()[0] << "; " << out->dims()[1] << "; "
          << out->dims()[2] << "; " << out->dims()[3];

  int count = static_cast<int>(n * out_h * out_w);
  auto cu_stream = dev_ctx.stream();
  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, count);
  GridSampleCudaKernel<
      T><<<config.block_per_grid, config.thread_per_block, 0, cu_stream>>>(
      count,
      n,
      c,
      out_h,
      out_w,
      in_h,
      in_w,
      x.data<T>(),
      grid.data<T>(),
      output_data,
      enum_mode,
      enum_padding_mode,
      align_corners);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    grid_sample, GPU, ALL_LAYOUT, phi::GridSampleKernel, float, double) {}
