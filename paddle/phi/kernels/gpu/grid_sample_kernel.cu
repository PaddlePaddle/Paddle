#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/grid_sample_kernel.h"

#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/grid_sample_utils.h"

namespace phi {

template <typename T>
static __forceinline__ __device__ T Unnormalize(T coord,
                                                int size,
                                                bool align_corners) {
  if (align_corners) {
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    return ((coord + 1.f) * size - 1) / 2;
  }
}

template <typename T>
static __forceinline__ __device__ T ClipIndexes(T in, int max_value) {
  return min(static_cast<T>(max_value), max(in, static_cast<T>(0)));
}

template <typename T>
static __forceinline__ __device__ T ReflectIndexes(T in,
                                                   int twice_low,
                                                   int twice_high) {
  if (twice_low == twice_high) {
    return static_cast<T>(0);
  }
  T min = static_cast<T>(twice_low) / 2;
  T span = static_cast<T>(twice_high - twice_low) / 2;
  in = fabs(in - min);
  T extra = fmod(in, span);
  int flips = static_cast<int>(floor(in / span));
  if (flips % 2 == 0) {
    return extra + min;
  } else {
    return span - extra + min;
  }
}

template <typename T>
static __forceinline__ __device__ T ComputePositions(T coord,
                                                     int size,
                                                     PaddingMode padding_mode,
                                                     bool align_corners) {
  coord = Unnormalize<T>(coord, size, align_corners);
  if (padding_mode == PaddingMode::border) {
    coord = ClipIndexes(coord, size - 1);
  } else if (padding_mode == PaddingMode::reflect) {
    if (align_corners) {
      coord = ReflectIndexes(coord, 0, 2 * (size - 1));
    } else {
      coord = ReflectIndexes(coord, -1, 2 * size - 1);
    }
    coord = ClipIndexes(coord, size - 1);
  }
  return coord;
}

template <typename T>
__global__ void GridSampleCudaKernel(const int nthreads,
                                     int n,
                                     int out_c,
                                     int out_h,
                                     int out_w,
                                     int in_h,
                                     int in_w,
                                     const T* input,
                                     const T* grid,
                                     T* output,
                                     const Mode mode,
                                     const PaddingMode padding_mode,
                                     bool align_corners) {
  int inp_sN = out_c * in_h * in_w;

  int inp_sC = in_h * in_w;
  int inp_sH = in_w;
  int inp_sW = 1;
  int grid_sN = out_h * out_w * 2;
  int grid_sH = out_w * 2;
  int grid_sW = 2;
  int grid_sCoor = 1;
  int out_sN = out_c * out_h * out_w;
  int out_sC = out_h * out_w;
  int out_sH = out_w;
  int out_sW = 1;
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_w;
    const int h = (index / out_w) % out_h;
    const int n = index / (out_h * out_w);
    const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];

    ix = ComputePositions(ix, in_w, padding_mode, align_corners);
    iy = ComputePositions(iy, in_h, padding_mode, align_corners);
    if (mode == Mode::bilinear) {
      int ix_nw = static_cast<int>(floor(ix));
      int iy_nw = static_cast<int>(floor(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      T nw = (ix_se - ix) * (iy_se - iy);
      T ne = (ix - ix_sw) * (iy_sw - iy);
      T sw = (ix_ne - ix) * (iy - iy_ne);
      T se = (ix - ix_nw) * (iy - iy_nw);

      auto inp_offset_NC = n * inp_sN;

      auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < out_c;
           ++c, inp_offset_NC += inp_sC, out_ptr_NCHW += out_sC) {
        *out_ptr_NCHW = static_cast<T>(0);
        if (InBounds(iy_nw, ix_nw, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_nw * inp_sH + ix_nw * inp_sW] * nw;
        }
        if (InBounds(iy_ne, ix_ne, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_ne * inp_sH + ix_ne * inp_sW] * ne;
        }
        if (InBounds(iy_sw, ix_sw, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_sw * inp_sH + ix_sw * inp_sW] * sw;
        }
        if (InBounds(iy_se, ix_se, in_h, in_w)) {
          *out_ptr_NCHW +=
              input[inp_offset_NC + iy_se * inp_sH + ix_se * inp_sW] * se;
        }
      }
    } else if (mode == Mode::nearest) {
      int ix_nearest = static_cast<int>(std::nearbyint(ix));
      int iy_nearest = static_cast<int>(std::nearbyint(iy));
      auto inp_offset_NC = n * inp_sN;
      auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < out_c;
           ++c, inp_offset_NC += inp_sC, out_ptr_NCHW += out_sC) {
        if (InBounds(iy_nearest, ix_nearest, in_h, in_w)) {
          *out_ptr_NCHW =
              input[inp_offset_NC + iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr_NCHW = static_cast<T>(0);
        }
      }
    }
  }
}

template <typename T>
__global__ void GridSample3DCudaKernel(const int nthreads,
                                       int out_c,
                                       int out_d,
                                       int out_h,
                                       int out_w,
                                       int in_d,
                                       int in_h,
                                       int in_w,
                                       const T* input,
                                       const T* grid,
                                       T* output,
                                       const Mode interpolation_mode,
                                       const PaddingMode padding_mode,
                                       bool align_corners) {
  int inp_sW = 1;
  int inp_sH = in_w;
  int inp_sD = in_h * in_w;
  int inp_sC = in_d * inp_sD;
  int inp_sN = out_c * inp_sC;

  int grid_sCoor = 1;
  int grid_sW = 3;
  int grid_sH = out_w * grid_sW;
  int grid_sD = out_h * grid_sH;
  int grid_sN = out_d * grid_sD;

  int out_sW = 1;
  int out_sH = out_w;
  int out_sD = out_h * out_w;
  int out_sC = out_d * out_sD;
  int out_sN = out_c * out_sC;

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, int) {
    const int w = index % out_w;
    const int h = (index / out_w) % out_h;
    const int d = (index / (out_h * out_w)) % out_d;
    const int n = index / (out_d * out_h * out_w);
    const int grid_offset =
        n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;
    // get the corresponding input x, y, z co-ordinates from grid
    T ix = grid[grid_offset];
    T iy = grid[grid_offset + grid_sCoor];
    T iz = grid[grid_offset + 2 * grid_sCoor];
    ix = ComputePositions(ix, in_w, padding_mode, align_corners);
    iy = ComputePositions(iy, in_h, padding_mode, align_corners);
    iz = ComputePositions(iz, in_d, padding_mode, align_corners);
    if (interpolation_mode == Mode::bilinear) {
      // get corner pixel values from (x, y, z)
      // for 4d, we used north-east-south-west
      // for 5d, we add top-bottom
      int ix_tnw = static_cast<int>(std::floor(ix));
      int iy_tnw = static_cast<int>(std::floor(iy));
      int iz_tnw = static_cast<int>(std::floor(iz));

      int ix_tne = ix_tnw + 1;
      int iy_tne = iy_tnw;
      int iz_tne = iz_tnw;

      int ix_tsw = ix_tnw;
      int iy_tsw = iy_tnw + 1;
      int iz_tsw = iz_tnw;

      int ix_tse = ix_tnw + 1;
      int iy_tse = iy_tnw + 1;
      int iz_tse = iz_tnw;

      int ix_bnw = ix_tnw;
      int iy_bnw = iy_tnw;
      int iz_bnw = iz_tnw + 1;

      int ix_bne = ix_tnw + 1;
      int iy_bne = iy_tnw;
      int iz_bne = iz_tnw + 1;

      int ix_bsw = ix_tnw;
      int iy_bsw = iy_tnw + 1;
      int iz_bsw = iz_tnw + 1;

      int ix_bse = ix_tnw + 1;
      int iy_bse = iy_tnw + 1;
      int iz_bse = iz_tnw + 1;

      // get surfaces to each neighbor:
      T tnw = (ix_bse - ix) * (iy_bse - iy) * (iz_bse - iz);
      T tne = (ix - ix_bsw) * (iy_bsw - iy) * (iz_bsw - iz);
      T tsw = (ix_bne - ix) * (iy - iy_bne) * (iz_bne - iz);
      T tse = (ix - ix_bnw) * (iy - iy_bnw) * (iz_bnw - iz);
      T bnw = (ix_tse - ix) * (iy_tse - iy) * (iz - iz_tse);
      T bne = (ix - ix_tsw) * (iy_tsw - iy) * (iz - iz_tsw);
      T bsw = (ix_tne - ix) * (iy - iy_tne) * (iz - iz_tne);
      T bse = (ix - ix_tnw) * (iy - iy_tnw) * (iz - iz_tnw);

      auto inp_ptr_NC = input + n * inp_sN;
      auto out_ptr_NCDHW =
          output + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
      for (int c = 0; c < out_c;
           ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
        *out_ptr_NCDHW = static_cast<T>(0);
        if (InBounds3D(iz_tnw, iy_tnw, ix_tnw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW] *
              tnw;
        }
        if (InBounds3D(iz_tne, iy_tne, ix_tne, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW] *
              tne;
        }
        if (InBounds3D(iz_tsw, iy_tsw, ix_tsw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW] *
              tsw;
        }
        if (InBounds3D(iz_tse, iy_tse, ix_tse, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW] *
              tse;
        }
        if (InBounds3D(iz_bnw, iy_bnw, ix_bnw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW] *
              bnw;
        }
        if (InBounds3D(iz_bne, iy_bne, ix_bne, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW] *
              bne;
        }
        if (InBounds3D(iz_bsw, iy_bsw, ix_bsw, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW] *
              bsw;
        }
        if (InBounds3D(iz_bse, iy_bse, ix_bse, in_d, in_h, in_w)) {
          *out_ptr_NCDHW +=
              inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW] *
              bse;
        }
      }
    } else if (interpolation_mode == Mode::nearest) {
      int ix_nearest = static_cast<int>(std::round(ix));
      int iy_nearest = static_cast<int>(std::round(iy));
      int iz_nearest = static_cast<int>(std::round(iz));

      // assign nearest neighor pixel value to output pixel
      auto inp_ptr_NC = input + n * inp_sN;
      auto out_ptr_NCDHW =
          output + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
      for (int c = 0; c < out_c;
           ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
        if (InBounds3D(iz_nearest, iy_nearest, ix_nearest, in_d, in_h, in_w)) {
          *out_ptr_NCDHW =
              inp_ptr_NC[iz_nearest * inp_sD + iy_nearest * inp_sH +
                         ix_nearest * inp_sW];
        } else {
          *out_ptr_NCDHW = static_cast<T>(0);
        }
      }
    }
  }
}

template <typename T, typename Context>
void GridSampleKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      const DenseTensor& grid,
                      const std::string& mode,
                      const std::string& padding_mode,
                      bool align_corners,
                      DenseTensor* out) {
  PaddingMode enum_padding_mode;
  Mode enum_mode;
  if (padding_mode == "border") {
    enum_padding_mode = PaddingMode::border;
  } else if (padding_mode == "reflection") {
    enum_padding_mode = PaddingMode::reflect;
  } else {
    enum_padding_mode = PaddingMode::zeros;
  }

  if (mode == "nearest") {
    enum_mode = Mode::nearest;
  } else {
    enum_mode = Mode::bilinear;
  }

  if (x.dims().size() == 4) {
    const int n = grid.dims()[0];
    const int out_h = grid.dims()[1];
    const int out_w = grid.dims()[2];
    const int c = x.dims()[1];
    const int in_h = x.dims()[2];
    const int in_w = x.dims()[3];
    VLOG(3) << "n: " << n << "; c: " << c << "; out_h: " << out_h
            << "; out_w: " << out_w;

    auto* output_data = dev_ctx.template Alloc<T>(out);
    VLOG(3) << "out dims: " << out->dims()[0] << "; " << out->dims()[1] << "; "
            << out->dims()[2] << "; " << out->dims()[3];

    int count = static_cast<int>(n * out_h * out_w);
    auto cu_stream = dev_ctx.stream();
    backends::gpu::GpuLaunchConfig config =
        backends::gpu::GetGpuLaunchConfig1D(dev_ctx, count);
    GridSampleCudaKernel<T>
        <<<config.block_per_grid, config.thread_per_block, 0, cu_stream>>>(
            count,
            n,
            c,
            out_h,
            out_w,
            in_h,
            in_w,
            x.data<T>(),
            grid.data<T>(),
            output_data,
            enum_mode,
            enum_padding_mode,
            align_corners);
  } else {
    const int n = grid.dims()[0];
    const int out_d = grid.dims()[1];
    const int out_h = grid.dims()[2];
    const int out_w = grid.dims()[3];
    const int c = x.dims()[1];
    const int in_d = x.dims()[2];
    const int in_h = x.dims()[3];
    const int in_w = x.dims()[4];

    VLOG(3) << "n: " << n << "; c: " << c << "; out_d: " << out_d
            << "; out_h: " << out_h << "; out_w: " << out_w;

    auto* output_data = dev_ctx.template Alloc<T>(out);
    VLOG(3) << "out dims: " << out->dims()[0] << "; " << out->dims()[1] << "; "
            << out->dims()[2] << "; " << out->dims()[3] << "; "
            << out->dims()[4];

    int count = static_cast<int>(n * out_d * out_h * out_w);
    auto cu_stream = dev_ctx.stream();
    backends::gpu::GpuLaunchConfig config =
        backends::gpu::GetGpuLaunchConfig1D(dev_ctx, count);
    GridSample3DCudaKernel<T>
        <<<config.block_per_grid, config.thread_per_block, 0, cu_stream>>>(
            count,
            c,
            out_d,
            out_h,
            out_w,
            in_d,
            in_h,
            in_w,
            x.data<T>(),
            grid.data<T>(),
            output_data,
            enum_mode,
            enum_padding_mode,
            align_corners);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(
    grid_sample, GPU, ALL_LAYOUT, phi::GridSampleKernel, float, double) {}
