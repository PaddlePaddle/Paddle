//   Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/elementwise_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/copy_kernel.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/gpu/elementwise_grad.h"
#include "paddle/phi/kernels/impl/elementwise_grad_kernel_impl.h"

namespace phi {

template <typename T>
void AddGradFunc(const GPUContext& dev_ctx,
                 const DenseTensor& x,
                 const DenseTensor& y,
                 const DenseTensor& out,
                 const DenseTensor& dout,
                 DenseTensor* dx,
                 DenseTensor* dy,
                 int axis = -1) {
  if (dx != nullptr && dy != nullptr && (dx->dims() == dy->dims())) {
    ElementwiseAddGrad<T>(dev_ctx, x, y, out, dout, dx, dy);
  } else {
    DefaultElementwiseAddGrad<T>(dev_ctx, x, y, out, dout, dx, dy, axis);
  }
}

template <typename T, typename Context>
void AddGradKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& y,
                   const DenseTensor& dout,
                   int axis,
                   DenseTensor* dx,
                   DenseTensor* dy) {
  phi::AddGradImpl<T>(dev_ctx, x, y, dout, axis, dx, dy, AddGradFunc<T>);
}

template <typename T, typename Context>
void AddDoubleGradKernel(const Context& dev_ctx,
                         const DenseTensor& y,
                         paddle::optional<const DenseTensor&> ddx,
                         paddle::optional<const DenseTensor&> ddy,
                         const DenseTensor& dout,
                         int axis,
                         DenseTensor* ddout) {
  phi::AddDoubleGradImpl<T>(dev_ctx, y, ddx, ddy, dout, axis, ddout);
}

template <typename T, typename Context>
void AddTripleGradKernel(const Context& dev_ctx,
                         const DenseTensor& ddx,
                         const DenseTensor& ddy,
                         const DenseTensor& d_ddout,
                         int axis,
                         DenseTensor* d_ddx,
                         DenseTensor* d_ddy) {
  phi::AddGradImpl<T>(
      dev_ctx, ddx, ddy, d_ddout, axis, d_ddx, d_ddy, AddGradFunc<T>);
}

template <typename T, typename Context>
void SubtractGradKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& y,
                        const DenseTensor& dout,
                        int axis,
                        DenseTensor* dx,
                        DenseTensor* dy) {
  // skip out
  auto* out = &dout;
  if (dx != nullptr && dy != nullptr && (dx->dims() == dy->dims())) {
    elementwise_sub_grad<T>(dev_ctx, x, y, *out, dout, dx, dy);
  } else {
    default_elementwise_sub_grad<T>(dev_ctx, x, y, *out, dout, dx, dy, axis);
  }
}

template <typename T, typename Context>
void SubtractDoubleGradKernel(const Context& dev_ctx,
                              const DenseTensor& y,
                              paddle::optional<const DenseTensor&> ddx,
                              paddle::optional<const DenseTensor&> ddy,
                              const DenseTensor& dout,
                              int axis,
                              DenseTensor* ddout) {
  phi::SubtractDoubleGradImpl<T>(dev_ctx, y, ddx, ddy, dout, axis, ddout);
}

}  // namespace phi

PD_REGISTER_KERNEL(add_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(add_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddDoubleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(add_triple_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddTripleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(subtract_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SubtractGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(subtract_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SubtractDoubleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
