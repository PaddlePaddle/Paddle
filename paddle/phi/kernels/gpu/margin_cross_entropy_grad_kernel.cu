#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// old op include, fluid should be removed
#ifdef PADDLE_WITH_HIP
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#else
#include <hipcub/hipcub.hpp>
#endif

#include <vector>
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/impl/softmax_kernel_impl.h"
#include "paddle/phi/kernels/margin_cross_entropy_grad_kernel.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/ProcessGroup.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif
#include "paddle/phi/backends/gpu/gpu_context.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename Context>
void GetClassInterval(const gpuStream_t& stream,
                      const phi::Place& place,
                      const Context& dev_ctx,
                      const int rid,
                      const int rank,
                      const int nranks,
                      const int D,
                      DenseTensor* class_interval) {
  std::vector<int> shard_dim_vec(nranks + 1, 0);
  shard_dim_vec[rank + 1] = D;
  if (nranks <= 1) {
    paddle::framework::TensorFromVector(shard_dim_vec, dev_ctx, class_interval);
    return;
  }
#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  DenseTensor num_classes_per_device;
  paddle::framework::TensorFromVector(
      shard_dim_vec, dev_ctx, &num_classes_per_device);
  int* num_classes_per_device_ptr = num_classes_per_device.data<int>();

  auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();
  if (map->has(rid)) {
    // Use ProcessGroup
    paddle::distributed::ProcessGroup* pg = map->get(rid);
    std::vector<phi::DenseTensor> in_tensor;
    std::vector<phi::DenseTensor> out_tensor;
    in_tensor.push_back(num_classes_per_device);
    out_tensor.push_back(num_classes_per_device);

    paddle::distributed::AllreduceOptions opts;
    opts.reduce_op = paddle::distributed::ReduceOp::SUM;
    auto task = pg->AllReduce(in_tensor, out_tensor, opts);
    task->Wait();
  } else {
    const auto& comm =
        paddle::platform::NCCLCommContext::Instance().Get(rid, place);
    // use global calculate stream
    const auto calcu_stream =
        static_cast<GPUContext*>(
            paddle::platform::DeviceContextPool::Instance().Get(place))
            ->stream();

    PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::ncclAllReduce(
        num_classes_per_device_ptr,
        num_classes_per_device_ptr,
        num_classes_per_device.numel(),
        paddle::platform::ToNCCLDataType(paddle::framework::TransToProtoVarType(
            num_classes_per_device.dtype())),
        ncclSum,
        comm->comm(),
        calcu_stream));
  }

  class_interval->Resize({nranks + 1});
  auto class_interval_ptr = dev_ctx.template Alloc<int>(class_interval);

  size_t cub_temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum<int*, int*>(
      nullptr, cub_temp_storage_bytes, nullptr, nullptr, nranks + 1, stream);
  auto cub_temp_storage = paddle::memory::Alloc(place, cub_temp_storage_bytes);
  hipcub::DeviceScan::InclusiveSum<int*, int*>(cub_temp_storage->ptr(),
                                            cub_temp_storage_bytes,
                                            num_classes_per_device_ptr,
                                            class_interval_ptr,
                                            nranks + 1,
                                            stream);
  return;
#endif
}

template <typename T, typename IndexT>
__global__ void CalculateGrad(T* logits_grad,
                              const T* loss_grad,
                              const T* logits,
                              const IndexT* label,
                              const float margin1,
                              const float margin2,
                              const float scale,
                              const int rank,
                              const int64_t N,
                              const int64_t D,
                              const int* class_interval_ptr) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  int start_index = class_interval_ptr[rank];
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    auto col = i % D;
    if ((col + start_index) == label[row]) {
      logits_grad[i] = (logits_grad[i] - static_cast<T>(1.0)) * loss_grad[row];
      if (fabs(margin1 - 1.0) > 1e-8 || fabs(margin2) > 1e-8) {
        MPType dout = static_cast<MPType>(logits_grad[i]);
        MPType one = static_cast<MPType>(1.0f);
        MPType x = static_cast<MPType>(logits[i]);
        MPType m1 = static_cast<MPType>(margin1);
        MPType m2 = static_cast<MPType>(margin2);

        MPType d = m1 * sin(m1 * acos(x) + m2) / sqrt(one - x * x);
        logits_grad[i] = static_cast<T>(dout * d);
      }
    } else {
      logits_grad[i] *= loss_grad[row];
    }
    if (fabs(scale - 1.0) > 1e-8) {
      logits_grad[i] *= static_cast<T>(scale);
    }
  }
}

template <typename T, typename Context>
void MarginCrossEntropyGradKernel(const Context& dev_ctx,
                                  const DenseTensor& logits,
                                  const DenseTensor& label,
                                  const DenseTensor& softmax,
                                  const DenseTensor& loss_grad,
                                  bool return_softmax,
                                  int ring_id,
                                  int rank,
                                  int nranks,
                                  float margin1,
                                  float margin2,
                                  float margin3,
                                  float scale,
                                  DenseTensor* logits_grad) {
  const auto softmax_dims = softmax.dims();
  const int axis = softmax_dims.size() - 1;
  const int N = phi::funcs::SizeToAxis(axis, softmax_dims);
  const int D = phi::funcs::SizeFromAxis(axis, softmax_dims);

  if (return_softmax) {
    phi::Copy<Context>(
        dev_ctx, softmax, dev_ctx.GetPlace(), false, logits_grad);
  } else {
    logits_grad->ShareDataWith(softmax);
  }

  int blocks = NumBlocks(N * D);
  int threads = kNumCUDAThreads;
  const auto& label_type =
      paddle::framework::TransToProtoVarType(label.dtype());

  DenseTensor class_interval;
  GetClassInterval<T, Context>(dev_ctx.stream(),
                               dev_ctx.GetPlace(),
                               dev_ctx,
                               ring_id,
                               rank,
                               nranks,
                               D,
                               &class_interval);

  if (label_type == paddle::framework::proto::VarType::INT32) {
    typedef int32_t LabelT;
    CalculateGrad<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logits_grad->data<T>(),
                                                   loss_grad.data<T>(),
                                                   logits.data<T>(),
                                                   label.data<LabelT>(),
                                                   margin1,
                                                   margin2,
                                                   scale,
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  } else if (label_type == paddle::framework::proto::VarType::INT64) {
    typedef int64_t LabelT;
    CalculateGrad<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logits_grad->data<T>(),
                                                   loss_grad.data<T>(),
                                                   logits.data<T>(),
                                                   label.data<LabelT>(),
                                                   margin1,
                                                   margin2,
                                                   scale,
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(margin_cross_entropy_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::MarginCrossEntropyGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
