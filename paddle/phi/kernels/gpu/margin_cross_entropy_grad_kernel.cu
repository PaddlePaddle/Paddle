#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// old op include, fluid should be removed
#ifdef PADDLE_WITH_HIP
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#else
#include <hipcub/hipcub.hpp>
#endif

#include <vector>
#include "paddle/phi/core/dense_tensor.h"

#include "paddle/phi/api/include/tensor.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/kernels/cpu/reduce.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/impl/softmax_kernel_impl.h"
#include "paddle/phi/kernels/margin_cross_entropy_grad_kernel.h"
#include "paddle/phi/kernels/margin_cross_entropy_kernel.h"

#include "paddle/phi/backends/cpu/cpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/ProcessGroup.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif
// trace op include
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"

namespace phi {
using Tensor = DenseTensor;

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename Context>
void GetClassInterval(const gpuStream_t& stream,
                      const phi::Place& place,
                      const Context& ctx,
                      const int rid,
                      const int rank,
                      const int nranks,
                      const int D,
                      Tensor* class_interval) {
  std::vector<int> shard_dim_vec(nranks + 1, 0);
  shard_dim_vec[rank + 1] = D;
  if (nranks <= 1) {
    paddle::framework::TensorFromVector(shard_dim_vec, ctx, class_interval);
    return;
  }
}
template <typename T, typename IndexT>
__global__ void CalculateGrad(T* logits_grad,
                              const T* loss_grad,
                              const T* logits,
                              const IndexT* label,
                              const float margin1,
                              const float margin2,
                              const float scale,
                              const int rank,
                              const int64_t N,
                              const int64_t D,
                              const int* class_interval_ptr) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  int start_index = class_interval_ptr[rank];
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    auto col = i % D;
    if ((col + start_index) == label[row]) {
      logits_grad[i] = (logits_grad[i] - static_cast<T>(1.0)) * loss_grad[row];
      if (fabs(margin1 - 1.0) > 1e-8 || fabs(margin2) > 1e-8) {
        MPType dout = static_cast<MPType>(logits_grad[i]);
        MPType one = static_cast<MPType>(1.0f);
        MPType x = static_cast<MPType>(logits[i]);
        MPType m1 = static_cast<MPType>(margin1);
        MPType m2 = static_cast<MPType>(margin2);

        MPType d = m1 * sin(m1 * acos(x) + m2) / sqrt(one - x * x);
        logits_grad[i] = static_cast<T>(dout * d);
      }
    } else {
      logits_grad[i] *= loss_grad[row];
    }
    if (fabs(scale - 1.0) > 1e-8) {
      logits_grad[i] *= static_cast<T>(scale);
    }
  }
}

template <typename T, typename Context>
void MarginCrossEntropyGradKernel(const Context& dev_ctx,
                                  const DenseTensor& logits,
                                  const DenseTensor& label,
                                  const DenseTensor& softmax,
                                  const DenseTensor& loss_grad,
                                  bool return_softmax,
                                  int ring_id,
                                  int rank,
                                  int nranks,
                                  float margin1,
                                  float margin2,
                                  float margin3,
                                  float scale,
                                  DenseTensor* logit_grad) {
  const auto softmax_dims = softmax.dims();
  const int axis = softmax_dims.size() - 1;
  const int N = phi::funcs::SizeToAxis(axis, softmax_dims);
  const int D = phi::funcs::SizeFromAxis(axis, softmax_dims);

  if (return_softmax) {
    phi::Copy(dev_ctx, softmax, dev_ctx.GetPlace(), false, logit_grad);
  } else {
    logit_grad->ShareDataWith(softmax);
  }

  int blocks = NumBlocks(N * D);
  int threads = kNumCUDAThreads;
  const auto& label_type =
      paddle::framework::TransToProtoVarType(label.dtype());

  Tensor class_interval;
  GetClassInterval<T, Context>(dev_ctx.stream(),
                               dev_ctx.GetPlace(),
                               dev_ctx,
                               ring_id,
                               rank,
                               nranks,
                               D,
                               &class_interval);

  if (label_type == paddle::framework::proto::VarType::INT32) {
    typedef int32_t LabelT;
    CalculateGrad<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logit_grad->data<T>(),
                                                   loss_grad.data<T>(),
                                                   logits.data<T>(),
                                                   label.data<LabelT>(),
                                                   margin1,
                                                   margin2,
                                                   scale,
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  } else if (label_type == paddle::framework::proto::VarType::INT64) {
    typedef int64_t LabelT;
    CalculateGrad<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logit_grad->data<T>(),
                                                   loss_grad.data<T>(),
                                                   logits.data<T>(),
                                                   label.data<LabelT>(),
                                                   margin1,
                                                   margin2,
                                                   scale,
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(margin_cross_entropy_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::MarginCrossEntropyGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
