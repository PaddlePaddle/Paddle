#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/arg_min_max_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

#include "paddle/fluid/framework/data_type.h"
#include "paddle/phi/core/ddim.h"
#define WARP_SIZE 32
#define BLOCK_SIZE 1024

namespace phi {

// shared memory
template <typename T>
struct SharedMemory {
  __device__ T* getPointer() { return NULL; }
};

template <>
struct SharedMemory<phi::dtype::float16> {
  __device__ phi::dtype::float16* getPointer() {
    extern __shared__ phi::dtype::float16 s_fp16[];
    return s_fp16;
  }
};

template <>
struct SharedMemory<float> {
  __device__ float* getPointer() {
    extern __shared__ float s_float[];
    return s_float;
  }
};

template <>
struct SharedMemory<double> {
  __device__ double* getPointer() {
    extern __shared__ double s_double[];
    return s_double;
  }
};

template <>
struct SharedMemory<int32_t> {
  __device__ int32_t* getPointer() {
    extern __shared__ int32_t s_int32_t[];
    return s_int32_t;
  }
};

template <>
struct SharedMemory<int64_t> {
  __device__ int64_t* getPointer() {
    extern __shared__ int64_t s_int64_t[];
    return s_int64_t;
  }
};

template <>
struct SharedMemory<int16_t> {
  __device__ int16_t* getPointer() {
    extern __shared__ int16_t s_int16_t[];
    return s_int16_t;
  }
};

template <>
struct SharedMemory<int8_t> {
  __device__ int8_t* getPointer() {
    extern __shared__ int8_t s_int8_t[];
    return s_int8_t;
  }
};

template <typename T>
struct CompMinOp {
  __device__ __forceinline__ T initial() {
    return static_cast<T>(std::numeric_limits<T>::max());
  }
  __device__ __forceinline__ bool operator()(const T a, const T b) const {
    return b < a;
  }
};

template <typename T>
struct CompMaxOp {
  __device__ __forceinline__ T initial() {
    return static_cast<T>(std::numeric_limits<T>::lowest());
  }
  __device__ __forceinline__ bool operator()(const T a, const T b) const {
    return a < b;
  }
};

template <typename T, typename IndType, typename CompOp>
__device__ void ArgWraper(T* values,
                          unsigned int* idx,
                          CompOp comp_op,
                          const unsigned int res_diff) {
  for (int stride = WARP_SIZE; stride > 0; stride >>= 1) {
    if (stride < res_diff &&
        comp_op(values[threadIdx.x], values[threadIdx.x + stride])) {
      values[threadIdx.x] = values[threadIdx.x + stride];
      idx[threadIdx.x] = idx[threadIdx.x + stride];
    }
  }
}

template <typename T, typename IndType, typename CompOp>
__global__ void ArgCudaKernel(const unsigned int length,
                              T* d_values,
                              IndType* d_index,
                              IndType* out,
                              const int64_t out_offset,
                              CompOp comp_op,
                              bool is_init) {
  unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int border = length >> 1;
  SharedMemory<T> shared;
  T* s_arg_values = shared.getPointer();
  s_arg_values[threadIdx.x] = comp_op.initial();
  if (tidx > border) return;

  const unsigned int res_diff = length - tidx;
  unsigned int* s_arg_idx = (unsigned int*)&s_arg_values[blockDim.x];
  unsigned int arg_id = is_init ? tidx : d_index[tidx];
  T arg_value = d_values[tidx];

  unsigned compare_idx = border + tidx;
  if (border < res_diff and comp_op(arg_value, d_values[compare_idx])) {
    arg_id = compare_idx;
    arg_value = d_values[arg_id];
  }

  s_arg_values[threadIdx.x] = arg_value;
  s_arg_idx[threadIdx.x] = arg_id;

  for (border = blockDim.x >> 1; border > 32; border >>= 1) {
    if (threadIdx.x > border) return;

    __syncthreads();
    compare_idx = threadIdx.x + border;  // within this block
    if (border < res_diff && compare_idx < blockDim.x &&
        comp_op(arg_value, s_arg_values[compare_idx])) {
      arg_value = s_arg_values[compare_idx];
      arg_id = s_arg_idx[compare_idx];
    }

    s_arg_values[threadIdx.x] = arg_value;
    s_arg_idx[threadIdx.x] = arg_id;
  }

  if (threadIdx.x < 32)
    ArgWraper<T, CompOp>(s_arg_values, s_arg_idx, comp_op, res_diff);

  if (threadIdx.x == 0) {
    d_values[blockIdx.x] = s_arg_values[0];
    d_index[blockIdx.x] = static_cast<IndType>(s_arg_idx[0]);
    out[out_offset] = d_index[blockIdx.x];
  }
}

template <typename Context, typename T, typename IndType, typename CompOp>
void ArgCUDAImpl(const Context& dev_ctx,
                 T* in,
                 int64_t length,
                 IndType* out,
                 CompOp comp_op,
                 const int64_t out_offset) {  //--->h_index (arg_max)
  int grid_size = std::ceil(length / float(BLOCK_SIZE) / 2);

  DenseTensor d_index;  // init d_index
  d_index.Resize(phi::make_ddim({grid_size}));
  dev_ctx.template Alloc<IndType>(&d_index);
  IndType* d_index_ptr = d_index.data<IndType>();

  const unsigned int s_mem_size =
      BLOCK_SIZE * (sizeof(T) + sizeof(unsigned int));
  ArgCudaKernel<T, IndType, CompOp><<<grid_size, BLOCK_SIZE, s_mem_size>>>(
      length, in, d_index_ptr, out, out_offset, comp_op, true);

  while (grid_size > 1) {
    length = grid_size;
    grid_size = std::ceil(length / float(BLOCK_SIZE) / 2);
    ArgCudaKernel<T, IndType, CompOp><<<grid_size, BLOCK_SIZE, s_mem_size>>>(
        length, in, d_index_ptr, out, out_offset, comp_op, false);
  }
}

template <typename Context, typename T, typename CompOp>
struct VisitDataCudaArgMinMaxFunctor {
  const Context& dev_ctx;
  const DenseTensor& x;
  int64_t axis;
  bool keepdims;
  bool flatten;
  CompOp comp_op;
  DenseTensor* out;

  explicit VisitDataCudaArgMinMaxFunctor(const Context& dev_ctx,
                                         const DenseTensor& x,
                                         int64_t axis,
                                         bool keepdims,
                                         bool flatten,
                                         CompOp comp_op,
                                         DenseTensor* out)
      : dev_ctx(dev_ctx),
        x(x),
        axis(axis),
        keepdims(keepdims),
        flatten(flatten),
        comp_op(comp_op),
        out(out) {}

  template <typename IndType>
  void apply() const {
    phi::DDim x_dims = x.dims();
    auto rank = x_dims.size();

    DenseTensor input;
    IndType* out_ptr = dev_ctx.template Alloc<IndType>(out);

    if (flatten || rank == 1) {
      input.Resize(phi::make_ddim({x.numel()}));
      T* input_ptr = dev_ctx.template Alloc<T>(&input);
      phi::Copy(dev_ctx, x, dev_ctx.GetPlace(), false, &input);
      ArgCUDAImpl<Context, T, IndType, CompOp>(
          dev_ctx, input_ptr, x.numel(), out_ptr, comp_op, 0);
    } else {
      int new_axis = axis;
      if (axis < 0) new_axis += rank;

      std::vector<int> perm;
      int64_t pre_dim = 1;
      DDim permed_shape(x_dims);
      for (int64_t i = 0; i < rank; i++) {
        if (i != axis) {
          perm.push_back(i);
          permed_shape[i] = x_dims[i];
          pre_dim *= x_dims[i];
        }
      }
      perm.push_back(new_axis);
      permed_shape[rank - 1] = x_dims[new_axis];
      int64_t post_dim = x_dims[new_axis];

      input.Resize(permed_shape);
      T* input_ptr = dev_ctx.template Alloc<T>(&input);
      funcs::TransCompute<Context, T>(rank, dev_ctx, x, &input, perm);

      int grid_size = std::ceil(post_dim / float(BLOCK_SIZE) / 2);
      DenseTensor d_index;  // init d_index
      d_index.Resize(phi::make_ddim({grid_size}));
      IndType* d_index_ptr = dev_ctx.template Alloc<IndType>(&d_index);
      const unsigned int s_mem_size =
          BLOCK_SIZE * (sizeof(T) + sizeof(unsigned int));

      for (int64_t i = 0; i < pre_dim; i++) {
        int64_t pos = static_cast<int64_t>(i * post_dim);
        ArgCUDAImpl<Context, T, IndType, CompOp>(
            dev_ctx, input_ptr + pos, post_dim, out_ptr, comp_op, i);
      }
    }
    out->Resize(out->dims());
  }
};

template <typename Context, typename T, typename CompOp>
void ArgMinMaxOpCUDAKernel(const Context& dev_ctx,
                           const DenseTensor& x,
                           const Scalar& axis,
                           bool keepdims,
                           bool flatten,
                           int dtype,
                           CompOp comp_op,
                           DenseTensor* out) {
  if (dtype < 0) {
    paddle::framework::VisitDataTypeTiny(
        static_cast<paddle::framework::proto::VarType::Type>(
            paddle::framework::proto::VarType::INT64),
        VisitDataCudaArgMinMaxFunctor<Context, T, CompOp>(
            dev_ctx, x, axis.to<int64_t>(), keepdims, flatten, comp_op, out));
    return;
  }
  paddle::framework::VisitDataTypeTiny(
      static_cast<paddle::framework::proto::VarType::Type>(dtype),
      VisitDataCudaArgMinMaxFunctor<Context, T, CompOp>(
          dev_ctx, x, axis.to<int64_t>(), keepdims, flatten, comp_op, out));
}

template <typename T, typename Context>
void ArgMinKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const Scalar& axis,
                  bool keepdims,
                  bool flatten,
                  int dtype,
                  DenseTensor* out) {
  ArgMinMaxOpCUDAKernel<Context, T, CompMinOp<T>>(
      dev_ctx, x, axis, keepdims, flatten, dtype, CompMinOp<T>(), out);
}

template <typename T, typename Context>
void ArgMaxKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  const Scalar& axis,
                  bool keepdims,
                  bool flatten,
                  int dtype,
                  DenseTensor* out) {
  ArgMinMaxOpCUDAKernel<Context, T, CompMaxOp<T>>(
      dev_ctx, x, axis, keepdims, flatten, dtype, CompMaxOp<T>(), out);
}

}  // namespace phi

PD_REGISTER_KERNEL(arg_min,
                   GPU,
                   ALL_LAYOUT,
                   phi::ArgMinKernel,
                   phi::dtype::float16,
                   float,
                   double,
                   int32_t,
                   int64_t,
                   int16_t,
                   uint8_t) {}

PD_REGISTER_KERNEL(arg_max,
                   GPU,
                   ALL_LAYOUT,
                   phi::ArgMaxKernel,
                   phi::dtype::float16,
                   float,
                   double,
                   int32_t,
                   int64_t,
                   int16_t,
                   uint8_t) {}
