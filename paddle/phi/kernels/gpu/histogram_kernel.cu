#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/histogram_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/eigen/eigen_function.h"

namespace phi {

using IndexType = int64_t;
using paddle::platform::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename IndexType>
__device__ static IndexType GetBin(T input_value,
                                   T min_value,
                                   T max_value,
                                   int64_t nbins) {
  IndexType bin = static_cast<int>((input_value - min_value) * nbins /
                                   (max_value - min_value));
  IndexType output_index = bin < nbins - 1 ? bin : nbins - 1;
  return output_index;
}

template <typename T, typename IndexType>
__global__ void KernelHistogram(const T* input,
                                const int total_elements,
                                const int64_t nbins,
                                const T min_value,
                                const T max_value,
                                int64_t* output) {
  extern __shared__ int64_t buf_hist[];
  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    buf_hist[i] = 0;
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(input_index, total_elements) {
    // const IndexType input_index = threadIdx.x + blockIdx.x * blockDim.x;
    const auto input_value = input[input_index];
    if (input_value >= min_value && input_value <= max_value) {
      const IndexType output_index =
          GetBin<T, IndexType>(input_value, min_value, max_value, nbins);
      paddle::platform::CudaAtomicAdd(&buf_hist[output_index], 1);
    }
  }
  __syncthreads();

  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    paddle::platform::CudaAtomicAdd(&output[i], buf_hist[i]);
  }
}

template <typename T, typename Context>
void HistogramKernel(const Context& dev_ctx,
                     const DenseTensor& input,
                     int64_t bins,
                     int min,
                     int max,
                     DenseTensor* output) {
  auto& nbins = bins;
  auto& minval = min;
  auto& maxval = max;

  const T* input_data = input.data<T>();
  const int input_numel = input.numel();

  int64_t* out_data = output->mutable_data<int64_t>(dev_ctx.GetPlace());
  phi::funcs::SetConstant<Context, int64_t>()(
      dev_ctx, output, static_cast<int64_t>(0));

  if (input_data == nullptr) return;

  T output_min = static_cast<T>(minval);
  T output_max = static_cast<T>(maxval);

  if (output_min == output_max) {
    auto input_x = phi::EigenVector<T>::Flatten(input);

    DenseTensor input_min_t, input_max_t;
    auto* input_min_data = input_min_t.mutable_data<T>({1}, dev_ctx.GetPlace());
    auto* input_max_data = input_max_t.mutable_data<T>({1}, dev_ctx.GetPlace());
    auto input_min_scala = phi::EigenScalar<T>::From(input_min_t);
    auto input_max_scala = phi::EigenScalar<T>::From(input_max_t);

    auto* place = dev_ctx.eigen_device();
    input_min_scala.device(*place) = input_x.minimum();
    input_max_scala.device(*place) = input_x.maximum();

    DenseTensor input_min_cpu, input_max_cpu;
    paddle::framework::TensorCopySync(
        input_min_t, phi::CPUPlace(), &input_min_cpu);
    paddle::framework::TensorCopySync(
        input_max_t, phi::CPUPlace(), &input_max_cpu);

    output_min = input_min_cpu.data<T>()[0];
    output_max = input_max_cpu.data<T>()[0];
  }
  if (output_min == output_max) {
    output_min = output_min - 1;
    output_max = output_max + 1;
  }

  PADDLE_ENFORCE_EQ((std::isinf(static_cast<float>(output_min)) ||
                     std::isnan(static_cast<float>(output_max)) ||
                     std::isinf(static_cast<float>(output_min)) ||
                     std::isnan(static_cast<float>(output_max))),
                    false,
                    phi::errors::OutOfRange("range of min, max is not finite"));
  PADDLE_ENFORCE_GE(
      output_max,
      output_min,
      phi::errors::InvalidArgument(
          "max must be larger or equal to min. If min and max are both zero, "
          "the minimum and maximum values of the data are used. "
          "But received max is %d, min is %d",
          maxval,
          minval));

  auto stream = dev_ctx.stream();
  KernelHistogram<T, IndexType><<<GET_BLOCKS(input_numel),
                                  PADDLE_CUDA_NUM_THREADS,
                                  nbins * sizeof(int64_t),
                                  stream>>>(
      input_data, input_numel, nbins, output_min, output_max, out_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(histogram,
                   GPU,
                   ALL_LAYOUT,
                   phi::HistogramKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
