#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/histogram_kernel.h"

<<<<<<< HEAD
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
=======
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/eigen/eigen_function.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

using IndexType = int64_t;
<<<<<<< HEAD
using phi::PADDLE_CUDA_NUM_THREADS;
=======
using paddle::platform::PADDLE_CUDA_NUM_THREADS;
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename IndexType>
__device__ static IndexType GetBin(T input_value,
                                   T min_value,
                                   T max_value,
                                   int64_t nbins) {
  IndexType bin = static_cast<int>((input_value - min_value) * nbins /
                                   (max_value - min_value));
  IndexType output_index = bin < nbins - 1 ? bin : nbins - 1;
  return output_index;
}

template <typename T, typename IndexType>
__global__ void KernelHistogram(const T* input,
                                const int total_elements,
                                const int64_t nbins,
                                const T min_value,
                                const T max_value,
                                int64_t* output) {
  extern __shared__ int64_t buf_hist[];
  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    buf_hist[i] = 0;
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(input_index, total_elements) {
    // const IndexType input_index = threadIdx.x + blockIdx.x * blockDim.x;
    const auto input_value = input[input_index];
    if (input_value >= min_value && input_value <= max_value) {
      const IndexType output_index =
          GetBin<T, IndexType>(input_value, min_value, max_value, nbins);
<<<<<<< HEAD
      phi::CudaAtomicAdd(&buf_hist[output_index], 1);
=======
      paddle::platform::CudaAtomicAdd(&buf_hist[output_index], 1);
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
    }
  }
  __syncthreads();

  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
<<<<<<< HEAD
    phi::CudaAtomicAdd(&output[i], buf_hist[i]);
=======
    paddle::platform::CudaAtomicAdd(&output[i], buf_hist[i]);
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
  }
}

template <typename T, typename Context>
void HistogramKernel(const Context& dev_ctx,
                     const DenseTensor& input,
                     int64_t bins,
                     int min,
                     int max,
                     DenseTensor* output) {
  auto& nbins = bins;
  auto& minval = min;
  auto& maxval = max;

  const T* input_data = input.data<T>();
  const int input_numel = input.numel();

<<<<<<< HEAD
  int64_t* out_data = dev_ctx.template Alloc<int64_t>(output);
=======
  int64_t* out_data = output->mutable_data<int64_t>(dev_ctx.GetPlace());
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
  phi::funcs::SetConstant<Context, int64_t>()(
      dev_ctx, output, static_cast<int64_t>(0));

  if (input_data == nullptr) return;

  T output_min = static_cast<T>(minval);
  T output_max = static_cast<T>(maxval);

  if (output_min == output_max) {
    auto input_x = phi::EigenVector<T>::Flatten(input);

    DenseTensor input_min_t, input_max_t;
<<<<<<< HEAD
    input_min_t.Resize({1});
    input_max_t.Resize({1});
    auto* input_min_data = dev_ctx.template Alloc<T>(&input_min_t);
    auto* input_max_data = dev_ctx.template Alloc<T>(&input_max_t);
=======
    auto* input_min_data = input_min_t.mutable_data<T>({1}, dev_ctx.GetPlace());
    auto* input_max_data = input_max_t.mutable_data<T>({1}, dev_ctx.GetPlace());
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
    auto input_min_scala = phi::EigenScalar<T>::From(input_min_t);
    auto input_max_scala = phi::EigenScalar<T>::From(input_max_t);

    auto* place = dev_ctx.eigen_device();
    input_min_scala.device(*place) = input_x.minimum();
    input_max_scala.device(*place) = input_x.maximum();

    DenseTensor input_min_cpu, input_max_cpu;
    paddle::framework::TensorCopySync(
        input_min_t, phi::CPUPlace(), &input_min_cpu);
    paddle::framework::TensorCopySync(
        input_max_t, phi::CPUPlace(), &input_max_cpu);

    output_min = input_min_cpu.data<T>()[0];
    output_max = input_max_cpu.data<T>()[0];
  }
  if (output_min == output_max) {
    output_min = output_min - 1;
    output_max = output_max + 1;
  }

  PADDLE_ENFORCE_EQ((std::isinf(static_cast<float>(output_min)) ||
                     std::isnan(static_cast<float>(output_max)) ||
                     std::isinf(static_cast<float>(output_min)) ||
                     std::isnan(static_cast<float>(output_max))),
                    false,
                    phi::errors::OutOfRange("range of min, max is not finite"));
  PADDLE_ENFORCE_GE(
      output_max,
      output_min,
      phi::errors::InvalidArgument(
          "max must be larger or equal to min. If min and max are both zero, "
          "the minimum and maximum values of the data are used. "
          "But received max is %d, min is %d",
          maxval,
          minval));

  auto stream = dev_ctx.stream();
  KernelHistogram<T, IndexType><<<GET_BLOCKS(input_numel),
                                  PADDLE_CUDA_NUM_THREADS,
                                  nbins * sizeof(int64_t),
                                  stream>>>(
      input_data, input_numel, nbins, output_min, output_max, out_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(histogram,
                   GPU,
                   ALL_LAYOUT,
                   phi::HistogramKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
