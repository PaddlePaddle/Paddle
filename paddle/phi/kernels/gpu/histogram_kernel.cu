#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/histogram_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/eigen/eigen_function.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

using IndexType = int64_t;
using phi::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T, typename IndexType>
__device__ static IndexType GetBin(T input_value,
                                   T min_value,
                                   T max_value,
                                   int64_t nbins) {
  IndexType bin = static_cast<int>((input_value - min_value) * nbins /
                                   (max_value - min_value));
  IndexType output_index = bin < nbins - 1 ? bin : nbins - 1;
  return output_index;
}

template <typename T, typename IndexType>
__global__ void KernelHistogram(const T* input,
                                const int total_elements,
                                const int64_t nbins,
                                const T min_value,
                                const T max_value,
                                int64_t* output) {
  extern __shared__ int64_t buf_hist[];
  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    buf_hist[i] = 0;
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(input_index, total_elements) {
    // const IndexType input_index = threadIdx.x + blockIdx.x * blockDim.x;
    const auto input_value = input[input_index];
    if (input_value >= min_value && input_value <= max_value) {
      const IndexType output_index =
          GetBin<T, IndexType>(input_value, min_value, max_value, nbins);
      phi::CudaAtomicAdd(&buf_hist[output_index], 1);
    }
  }
  __syncthreads();

  for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
    phi::CudaAtomicAdd(&output[i], buf_hist[i]);
  }
}

template <typename T>
__global__ void KernelMinMax(const T* input,
                             const int total_elements,
                             T* output) {
  __shared__ T min_data;
  __shared__ T max_data;
  for (int i = threadIdx.x; i < 1; i += blockDim.x) {
    min_data = std::numeric_limits<T>::max();
    max_data = std::numeric_limits<T>::min();
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(index, total_elements) {
    const auto input_value = input[index];
    phi::CudaAtomicMin(&min_data, input_value);
    phi::CudaAtomicMax(&max_data, input_value);
  }
  __syncthreads();
  for (int i = threadIdx.x; i < 1; i += blockDim.x) {
    output[0] = min_data;
    output[1] = max_data;
  }
}

template <typename T, typename Context>
void HistogramKernel(const Context& dev_ctx,
                     const DenseTensor& input,
                     int64_t bins,
                     int min,
                     int max,
                     DenseTensor* output) {
  auto& nbins = bins;
  auto& minval = min;
  auto& maxval = max;

  const T* input_data = input.data<T>();
  const int input_numel = input.numel();

  int64_t* out_data = dev_ctx.template Alloc<int64_t>(output);
  phi::funcs::SetConstant<Context, int64_t>()(
      dev_ctx, output, static_cast<int64_t>(0));

  if (input_data == nullptr) return;

  T output_min = static_cast<T>(minval);
  T output_max = static_cast<T>(maxval);

  if (output_min == output_max) {
    DenseTensor min_max;
    min_max.Resize({2});
    auto* min_max_data = dev_ctx.template Alloc<T>(&min_max);
    KernelMinMax<T>
        <<<GET_BLOCKS(input_numel),
           PADDLE_CUDA_NUM_THREADS,
           0,
           dev_ctx.stream()>>>(input_data, input_numel, min_max_data);

    DenseTensor min_max_cpu;
    phi::Copy(dev_ctx, min_max, phi::CPUPlace(), true, &min_max_cpu);
    auto* min_max_cpu_data = min_max_cpu.data<T>();
    output_min = min_max_cpu_data[0];
    output_max = min_max_cpu_data[1];
  }
  if (output_min == output_max) {
    output_min = output_min - 1;
    output_max = output_max + 1;
  }

  PADDLE_ENFORCE_EQ((std::isinf(static_cast<float>(output_min)) ||
                     std::isnan(static_cast<float>(output_max)) ||
                     std::isinf(static_cast<float>(output_min)) ||
                     std::isnan(static_cast<float>(output_max))),
                    false,
                    phi::errors::OutOfRange("range of min, max is not finite"));
  PADDLE_ENFORCE_GE(
      output_max,
      output_min,
      phi::errors::InvalidArgument(
          "max must be larger or equal to min. If min and max are both zero, "
          "the minimum and maximum values of the data are used. "
          "But received max is %d, min is %d",
          maxval,
          minval));

  auto stream = dev_ctx.stream();
  KernelHistogram<T, IndexType><<<GET_BLOCKS(input_numel),
                                  PADDLE_CUDA_NUM_THREADS,
                                  nbins * sizeof(int64_t),
                                  stream>>>(
      input_data, input_numel, nbins, output_min, output_max, out_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(histogram,
                   GPU,
                   ALL_LAYOUT,
                   phi::HistogramKernel,
                   float,
                   double,
                   int,
                   int64_t) {
  kernel->OutputAt(0).SetDataType(paddle::DataType::INT64);
}
