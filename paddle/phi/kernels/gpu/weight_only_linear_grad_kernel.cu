#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "paddle/phi/kernels/weight_only_linear_grad_kernel.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/datatype_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/matmul_kernel.h"

#if defined(PADDLE_WITH_CUTLASS)
#include "paddle/phi/kernels/fusion/cutlass/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"
#endif

namespace phi {

#if defined(PADDLE_WITH_CUTLASS)
template <typename T, int WeightBit>
struct FastWeightOnlyHalfConverter;

template <>
struct FastWeightOnlyHalfConverter<half, 8> {
  using Converter =
      cutlass::FastInterleavedAndBiasedNumericArrayConverter<cutlass::half_t,
                                                             uint8_t,
                                                             4>;
  static constexpr int kHalfLength = 4;
  static constexpr int kWeightOnlyLength = 4;

  __device__ static inline void convert(half halves[kHalfLength],
                                        uint8_t chars[kWeightOnlyLength],
                                        float scale) {
    *reinterpret_cast<Converter::result_type*>(halves) =
        Converter::convert(*reinterpret_cast<Converter::source_type*>(chars));
#pragma unroll
    for (int i = 0; i < kHalfLength; ++i) {
      float dequant_value = __half2float(halves[i]) * scale;
      halves[i] = __float2half_rn(dequant_value);
    }
  }
};

template <>
struct FastWeightOnlyHalfConverter<half, 4> {
  using Converter =
      cutlass::FastInterleavedAndBiasedNumericArrayConverter<cutlass::half_t,
                                                             cutlass::uint4b_t,
                                                             8>;
  static constexpr int kHalfLength = 8;
  static constexpr int kWeightOnlyLength = 4;

  __device__ static inline void convert(half halves[kHalfLength],
                                        uint8_t chars[kWeightOnlyLength],
                                        float scale) {
    *reinterpret_cast<Converter::result_type*>(halves) =
        Converter::convert(*reinterpret_cast<Converter::source_type*>(chars));
#pragma unroll
    for (int i = 0; i < kHalfLength; ++i) {
      float dequant_value = __half2float(halves[i]) * scale;
      halves[i] = __float2half_rn(dequant_value);
    }
  }
};

#if defined(PADDLE_CUDA_BF16)
template <>
struct FastWeightOnlyHalfConverter<__hip_bfloat16, 8> {
  using Converter = cutlass::FastInterleavedAndBiasedNumericArrayConverter<
      cutlass::bfloat16_t,
      uint8_t,
      4>;
  static constexpr int kHalfLength = 4;
  static constexpr int kWeightOnlyLength = 4;

  __device__ static inline void convert(__hip_bfloat16 halves[kHalfLength],
                                        uint8_t chars[kWeightOnlyLength],
                                        float scale) {
    *reinterpret_cast<Converter::result_type*>(halves) =
        Converter::convert(*reinterpret_cast<Converter::source_type*>(chars));
#pragma unroll
    for (int i = 0; i < kHalfLength; ++i) {
      float dequant_value = __bfloat162float(halves[i]) * scale;
      halves[i] = __float2bfloat16_rn(dequant_value);
    }
  }
};

template <>
struct FastWeightOnlyHalfConverter<__hip_bfloat16, 4> {
  using Converter = cutlass::FastInterleavedAndBiasedNumericArrayConverter<
      cutlass::bfloat16_t,
      cutlass::uint4b_t,
      8>;
  static constexpr int kHalfLength = 8;
  static constexpr int kWeightOnlyLength = 4;

  __device__ static inline void convert(__hip_bfloat16 halves[kHalfLength],
                                        uint8_t chars[kWeightOnlyLength],
                                        float scale) {
    *reinterpret_cast<Converter::result_type*>(halves) =
        Converter::convert(*reinterpret_cast<Converter::source_type*>(chars));
#pragma unroll
    for (int i = 0; i < kHalfLength; ++i) {
      float dequant_value = __bfloat162float(halves[i]) * scale;
      halves[i] = __float2bfloat16_rn(dequant_value);
    }
  }
};
#endif

template <typename T>
__global__ void int8_weight_only_dequant(const uint8_t* weight,
                                         const float* scale_list,
                                         T* output,
                                         const int n,
                                         const int k) {
  using Converter = FastWeightOnlyHalfConverter<T, 8>;
  AlignedVector<uint8_t, 16> vec_weight;
  T vec_weight_f16[16];
  AlignedVector<T, 16> vec_out;

  int warp_id = threadIdx.x / 32, lane_id = threadIdx.x % 32;
  int tile_id = blockIdx.x * blockDim.x / 32 + warp_id;
  // Every two rows of the original weights are interleaved into a row with
  // stride of 64, so if each thread processes 16 elements(for int8, we can use
  // ldg.128 to load weights), then every group of four adjacent threads will
  // alternately process two different row weights for example every 128
  // consecutive int8 elements [128*i, 128*(i+1)-1] of row N under interleave
  // layout, the first 64 are from [64*i, 64*(i+1)-1] of row 2N before
  // interleaving, and the last 64 are from [64*i, 64*(i+1)-1] of row 2N+1
  // before interleaving. So if each thread loads 16 int8 elements, then the
  // elements of the first four and last four threads of each 8 consecutive
  // threads will come from row 2N and row 2N+1 respectively before
  // interleaving.
  int row_id = tile_id * 2 + ((lane_id % 8) > 3 ? 1 : 0);
  weight += tile_id * k * 2;
  output += tile_id * k * 2;
  float scale = scale_list[row_id];
#pragma unroll
  for (int i = lane_id * 16; i < k * 2; i += 16 * 32) {
    Load<uint8_t, 16>(&weight[i], &vec_weight);
#pragma unroll
    for (int p = 0; p < 16; p += Converter::kHalfLength) {
      // The rearrangement here counteracts the effect of
      // cutlass::add_bias_and_interleave_int8s_inplace Input int8 data layout
      //      [elt_3  elt_1  elt_2  elt_0] (each elt occupies 8 bits)
      //
      // Converted fp16 data layout
      //      [elt_3  elt_2  elt_1  elt_0] (each elt occupies 16 bits)
      // vec_weight_f16[p] = static_cast<T>(static_cast<float>(vec_weight[p]) *
      // scale);
      // fast_cvt_4_packed_signed_i8s_to_2_half2s<T>()
      Converter::convert(vec_weight_f16 + p, &vec_weight[p], scale);
    }
#pragma unroll
    for (int p = 0; p < 16; ++p) {
      // The index remapping here is to counteracts the effect of
      // cutlass::permute_B_rows_for_mixed_gemm input 0 1 2 3 4 5 6 7 8 9 10 11
      // 12 13 14 15 weight 0 1 8 9 2 3 10 11 4 5 12 13 6 7 14 15
      // printf("vec_weight_f16%d:  %f", p, static_cast<float>(vec_weight_f16[4
      // * ((p % 8) / 2) + p % 2 + 2 * (p / 8)]));
      vec_out[p] = vec_weight_f16[4 * ((p % 8) / 2) + p % 2 + 2 * (p / 8)];
    }
    Store<T, 16>(vec_out, &output[i]);
  }
}

template <typename T>
__global__ void int4_weight_only_dequant(const uint8_t* weight,
                                         const float* scale_list,
                                         T* output,
                                         const int n,
                                         const int k) {
  using Converter = FastWeightOnlyHalfConverter<T, 4>;

  AlignedVector<uint8_t, 16> vec_weight;
  T vec_weight_f16[32];
  AlignedVector<T, 32> vec_out;

  int warp_id = threadIdx.x / 32, lane_id = threadIdx.x % 32;
  int tile_id = blockIdx.x * blockDim.x / 32 + warp_id;
  // Every two rows of the original weights are interleaved into a row with
  // stride of 64, so if each thread processes 16 elements(for int8, we can use
  // ldg.128 to load weights), then every group of four adjacent threads will
  // alternately process two different row weights for example every 128
  // consecutive int8 elements [128*i, 128*(i+1)-1] of row N under interleave
  // layout, the first 64 are from [64*i, 64*(i+1)-1] of row 2N before
  // interleaving, and the last 64 are from [64*i, 64*(i+1)-1] of row 2N+1
  // before interleaving. So if each thread loads 16 int8 elements, then the
  // elements of the first four and last four threads of each 8 consecutive
  // threads will come from row 2N and row 2N+1 respectively before
  // interleaving.
  int row_id = tile_id * 4 + ((lane_id % 8) / 2);
  weight += tile_id * k / 2 * 4;
  output += tile_id * k / 2 * 4 * 2;
  float scale = scale_list[row_id];
#pragma unroll
  for (int i = lane_id * 32; i < k * 4; i += 32 * 32) {
    Load<uint8_t, 16>(&weight[i / 2], &vec_weight);
#pragma unroll
    for (int p = 0; p < 32; p += Converter::kHalfLength) {
      // The rearrangement here counteracts the effect of
      // cutlass::add_bias_and_interleave_int4s_inplace Input int8 data layout
      //      [elt_7  elt_5  elt_3  elt_1  elt_6  elt_4  elt_2  elt_0] (each elt
      //      occupies 4 bits)
      //
      // Converted fp16 data layout
      //      [elt_7  elt_6  elt_5  elt_4  elt_3  elt_2  elt_1  elt_0] (each elt
      //      occupies 16 bits)
      // vec_weight_f16[p] =
      //     static_cast<T>(static_cast<float>(vec_weight[p]) * scale);
      Converter::convert(vec_weight_f16 + p, &vec_weight[p / 2], scale);
    }
#pragma unroll
    for (int p = 0; p < 32; ++p) {
      // The index remapping here is to counteracts the effect of
      // cutlass::permute_B_rows_for_mixed_gemm input 0 1 2 3 4 5 6 7 8 9 10 11
      // 12 13 14 15 ... 31 weight 0 1 8 9 16 17 24 25 2 3 10 11 18 19 26 27 4 5
      // 12 13 20 21 28 29 6 7 14 15 22 23 30 31
      vec_out[p] = vec_weight_f16[8 * ((p % 8) / 2) + p % 2 + 2 * (p / 8)];
    }
    Store<T, 32>(vec_out, &output[i]);
  }
}
#endif

template <typename T, typename Context>
void WeightOnlyLinearGradKernel(const Context& dev_ctx,
                                const DenseTensor& x,
                                const DenseTensor& weight,
                                const paddle::optional<DenseTensor>& bias,
                                const DenseTensor& weight_scale,
                                const DenseTensor& out_grad,
                                const std::string& weight_dtype,
                                DenseTensor* x_grad) {
#if defined(PADDLE_WITH_CUTLASS)
  using DataType = typename PDDataTypeTraits<T>::DataType;
  int n = weight_scale.dims()[0];
  int k = weight.dims()[1];
  dim3 block(512);
  dim3 grid(n / 32);
  auto stream = dev_ctx.stream();

  dev_ctx.template Alloc<T>(x_grad);
  DenseTensor weight_dequantized;
  weight_dequantized.Resize({{n, k}});
  dev_ctx.template Alloc<T>(&weight_dequantized);

  T* weight_dequantized_data = weight_dequantized.data<T>();

  if (weight_dtype == "int8") {
    int8_weight_only_dequant<DataType><<<grid, block, 0, stream>>>(
        reinterpret_cast<const uint8_t*>(weight.data<int8_t>()),
        weight_scale.data<float>(),
        reinterpret_cast<DataType*>(weight_dequantized_data),
        n,
        k);
  } else if (weight_dtype == "int4") {
    grid.x /= 2;
    int4_weight_only_dequant<DataType><<<grid, block, 0, stream>>>(
        reinterpret_cast<const uint8_t*>(weight.data<int8_t>()),
        weight_scale.data<float>(),
        reinterpret_cast<DataType*>(weight_dequantized_data),
        n,
        k);
  }
  MatmulKernel<T, Context>(
      dev_ctx, out_grad, weight_dequantized, false, false, x_grad);
#endif
}
}  // namespace phi

PD_REGISTER_KERNEL(weight_only_linear_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::WeightOnlyLinearGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
