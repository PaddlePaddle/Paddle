/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/matmul_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/complex.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/matmul_kernel_impl.h"

PD_REGISTER_KERNEL(matmul,
                   GPU,
                   ALL_LAYOUT,
                   phi::MatmulKernel,
                   float,
                   double,
                   int32_t,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>,
                   int8_t) {
  if (kernel_key.dtype() == phi::DataType::INT8) {
    kernel->OutputAt(0).SetDataType(phi::DataType::INT32);
  }
}

PD_REGISTER_KERNEL(matmul_with_flatten,
                   GPU,
                   ALL_LAYOUT,
                   phi::MatmulWithFlattenKernel,
                   float,
                   double,
                   phi::dtype::bfloat16,
                   phi::dtype::float16,
                   int8_t) {
  if (kernel_key.dtype() == phi::DataType::INT8) {
    kernel->OutputAt(0).SetDataType(phi::DataType::INT32);
  }
}
