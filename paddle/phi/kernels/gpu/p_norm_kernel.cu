#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/p_norm_kernel.h"

#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/funcs/p_norm_utils.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/gpu/reduce.h"

namespace phi {
template <typename T>
struct NonzeroFunctor {
  HOSTDEVICE explicit inline NonzeroFunctor() {}
  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(static_cast<double>(x) != 0);
  }
};

template <typename T>
struct AbsFunctor {
  HOSTDEVICE explicit inline AbsFunctor() {}
  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(inline_abs(x));
  }
};

template <typename T>
struct UnsignedPowFunctor {
  HOSTDEVICE explicit inline UnsignedPowFunctor(float porder) {
    this->porder = porder;
  }
  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(inline_pow(inline_abs(x), static_cast<T>(porder)));
  }
  float porder;
};

template <typename T, typename Context>
void PNormKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 float porder,
                 int axis,
                 float epsilon,
                 bool keepdim,
                 bool asvector,
                 DenseTensor* out) {
  auto* in_x = &x;
  auto* out_norm = out;
  T* norm = dev_ctx.template Alloc<T>(out);
  auto xdim = in_x->dims();
  std::vector<int64_t> axis_dims = {static_cast<int64_t>(axis)};
  std::vector<int> reduce_axis =
      funcs::details::GetReduceDim(axis_dims, xdim.size(), asvector);

  for (int i = 0; i < xdim.size(); i++) {
    PADDLE_ENFORCE_LT(0,
                      xdim[i],
                      errors::InvalidArgument(
                          "The dims of Input(X) should be greater than 0."));
  }

  using MT = typename dtype::MPTypeTrait<T>::Type;
  if (porder == 0) {
    phi::funcs::ReduceKernel<T, T, kps::AddFunctor, NonzeroFunctor<T>>(
        dev_ctx, *in_x, out_norm, NonzeroFunctor<T>(), reduce_axis);
  } else if (porder == INFINITY) {
    phi::funcs::ReduceKernel<T, T, kps::MaxFunctor, AbsFunctor<T>>(
        dev_ctx, *in_x, out_norm, AbsFunctor<T>(), reduce_axis);
  } else if (porder == -INFINITY) {
    phi::funcs::ReduceKernel<T, T, kps::MinFunctor, AbsFunctor<T>>(
        dev_ctx, *in_x, out_norm, AbsFunctor<T>(), reduce_axis);
  } else {
    phi::funcs::ReduceKernel<T, T, kps::AddFunctor, UnsignedPowFunctor<T>>(
        dev_ctx, *in_x, out_norm, UnsignedPowFunctor<T>(porder), reduce_axis);

    const DenseTensor* tmp_norm = out_norm;
    std::vector<const DenseTensor*> ins = {tmp_norm};
    std::vector<DenseTensor*> outs = {out_norm};
    phi::funcs::ElementwiseKernel<T>(
        dev_ctx, ins, &outs, UnsignedPowFunctor<T>(1. / porder));
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(p_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::PNormKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
