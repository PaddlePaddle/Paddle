#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/p_norm_kernel.h"

#include "paddle/fluid/operators/elementwise/elementwise_op_impl.cu.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/gpu/reduce.h"

namespace phi {

template <typename T>
__device__ __forceinline__ int sgn(T val) {
  return (T(0) < val) - (val < T(0));
}

__device__ __forceinline__ dtype::float16 inline_abs(dtype::float16 x) {
  return static_cast<dtype::float16>(abs(static_cast<float>(x)));
}

__device__ __forceinline__ dtype::bfloat16 inline_abs(dtype::bfloat16 x) {
  return static_cast<dtype::bfloat16>(abs(static_cast<float>(x)));
}

__device__ __forceinline__ float inline_abs(float x) { return abs(x); }
__device__ __forceinline__ double inline_abs(double x) { return abs(x); }

__device__ __forceinline__ int inline_sign(dtype::float16 x) {
  return sgn<dtype::float16>(x);
}
__device__ __forceinline__ int inline_sign(float x) { return sgn<float>(x); }
__device__ __forceinline__ int inline_sign(double x) { return sgn<double>(x); }

__device__ __forceinline__ dtype::float16 inline_pow(dtype::float16 base,
                                                     dtype::float16 exponent) {
  return static_cast<dtype::float16>(
      pow(static_cast<float>(base), static_cast<float>(exponent)));
}
__device__ __forceinline__ dtype::bfloat16 inline_pow(
    dtype::bfloat16 base, dtype::bfloat16 exponent) {
  return static_cast<dtype::bfloat16>(
      pow(static_cast<float>(base), static_cast<float>(exponent)));
}
__device__ __forceinline__ float inline_pow(float base, float exponent) {
  return pow(base, exponent);
}
__device__ __forceinline__ double inline_pow(double base, double exponent) {
  return pow(base, exponent);
}

template <typename T>
struct NonzeroFunctor {
  HOSTDEVICE explicit inline NonzeroFunctor() {}
  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(static_cast<double>(x) != 0);
  }
};

template <typename T>
struct AbsFunctor {
  HOSTDEVICE explicit inline AbsFunctor() {}
  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(inline_abs(x));
  }
};

template <typename T>
struct UnsignedPowFunctor {
  HOSTDEVICE explicit inline UnsignedPowFunctor(float porder) {
    this->porder = porder;
  }
  HOSTDEVICE inline T operator()(const T x) const {
    return static_cast<T>(inline_pow(inline_abs(x), static_cast<T>(porder)));
  }
  float porder;
};

template <typename T, typename Context>
void PNormKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 float porder,
                 int axis,
                 float epsilon,
                 bool keepdim,
                 bool asvector,
                 DenseTensor* out) {
  auto* in_x = &x;
  auto* out_norm = out;
  T* norm = dev_ctx.template Alloc<T>(out);
  auto xdim = in_x->dims();
  std::vector<int64_t> axis_dims = {static_cast<int64_t>(axis)};
  std::vector<int> reduce_axis =
      funcs::details::GetReduceDim(axis_dims, xdim.size(), asvector);

  using MT = typename dtype::MPTypeTrait<T>::Type;
  if (porder == 0) {
    phi::funcs::ReduceKernel<T, T, kps::AddFunctor, NonzeroFunctor<T>>(
        dev_ctx, *in_x, out_norm, NonzeroFunctor<T>(), reduce_axis);
  } else if (porder == INFINITY) {
    phi::funcs::ReduceKernel<T, T, kps::MaxFunctor, AbsFunctor<T>>(
        dev_ctx, *in_x, out_norm, AbsFunctor<T>(), reduce_axis);
  } else if (porder == -INFINITY) {
    phi::funcs::ReduceKernel<T, T, kps::MinFunctor, AbsFunctor<T>>(
        dev_ctx, *in_x, out_norm, AbsFunctor<T>(), reduce_axis);
  } else {
    phi::funcs::ReduceKernel<T, T, kps::AddFunctor, UnsignedPowFunctor<T>>(
        dev_ctx, *in_x, out_norm, UnsignedPowFunctor<T>(porder), reduce_axis);

    const DenseTensor* tmp_norm = out_norm;
    std::vector<const DenseTensor*> ins = {tmp_norm};
    std::vector<DenseTensor*> outs = {out_norm};
    phi::funcs::ElementwiseKernel<T>(
        dev_ctx, ins, &outs, UnsignedPowFunctor<T>(1. / porder));
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(p_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::PNormKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
