#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/gumbel_softmax_kernel.h"
#include "paddle/phi/kernels/impl/gumbel_softmax_kernel_impl.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"

#if defined(__NVCC__) || defined(__HIPCC__)
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/fluid/framework/generator.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {
template <typename K, typename V>
using KeyValuePair = hipcub::KeyValuePair<K, V>;

template <typename T>
struct UniformCUDAGenerator {
  T min_, max_;
  unsigned int seed_;
  unsigned int offset_ = 0;
  HOSTDEVICE UniformCUDAGenerator(T min, T max, unsigned int seed)
      : min_(min), max_(max), seed_(seed) {}
  HOSTDEVICE UniformCUDAGenerator(T min,
                                  T max,
                                  unsigned int seed,
                                  unsigned int offset)
      : min_(min), max_(max), seed_(seed), offset_(offset) {}

  HOSTDEVICE T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n + offset_);
    return dist(rng);
  }
};

template <typename T, size_t BlockDim>
__global__ void OneHotCUDAKernel(const int64_t height,
                                 const int64_t width,
                                 const int64_t size_out_axis,
                                 const T init,
                                 const T* in,
                                 T* out) {
  typedef hipcub::BlockReduce<KeyValuePair<int, T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int64_t idx = blockIdx.x; idx < height; idx += gridDim.x) {
    KeyValuePair<int, T> kv_pair = {-1, init};
    int h = idx / size_out_axis;
    int w = idx % size_out_axis;
    hipcub::ArgMax reducer;
    for (int k = threadIdx.x; k < width; k += blockDim.x) {
      kv_pair = reducer(
          {k, in[h * width * size_out_axis + k * size_out_axis + w]}, kv_pair);
    }
    kv_pair = BlockReduce(temp_storage).Reduce(kv_pair, reducer);
    if (threadIdx.x == 0) {
      int index = static_cast<int>(kv_pair.key);
      out[h * width * size_out_axis + index * size_out_axis + w] = 1;
    }
    __syncthreads();
  }
}

template <typename T>
struct OneHotGenerator<GPUContext, T> {
  static void Transform(const GPUContext& ctx,
                        const DenseTensor& X,
                        DenseTensor* out,
                        int axis) {
    const int size_to_axis = funcs::SizeToAxis(axis, X.dims());
    const int size_from_axis = funcs::SizeFromAxis(axis, X.dims());
    const int size_out_axis = funcs::SizeOutAxis(axis, X.dims());
    constexpr int thread_size = 512;
    int64_t max_grid_dimx = ctx.GetCUDAMaxGridDimSize()[0];
    int64_t height = size_to_axis * size_out_axis;
    int block_size = height < max_grid_dimx ? height : max_grid_dimx;

    DenseTensor input_tensor;
    input_tensor.Resize(out->dims());
    ctx.template Alloc<T>(&input_tensor);
    paddle::framework::TensorCopy(*out, ctx.GetPlace(), &input_tensor);
    funcs::set_constant(ctx, out, 0.0);
    OneHotCUDAKernel<T,
                     thread_size><<<block_size, thread_size, 0, ctx.stream()>>>(
        height,
        size_from_axis / size_out_axis,
        size_out_axis,
        std::numeric_limits<T>::lowest(),
        input_tensor.data<T>(),
        out->data<T>());
  }
};

template <typename T>
__global__ void AddGumbelNoiseCUDAKernel(const T* input_data,
                                         T* output_data,
                                         T* noise,
                                         const float temperature,
                                         int64_t n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int step = blockDim.x * gridDim.x;
  for (int64_t i = index; i < n; i += step) {
    T gumbel_noise = -log(-log(noise[i]));
    output_data[i] = (gumbel_noise + input_data[i]) / temperature;
  }
}

template <typename T>
struct GumbleNoiseGenerator<GPUContext, T> {
  static void Transform(const GPUContext& ctx,
                        const T* input_data,
                        T* output_data,
                        int size_to_axis,
                        int size_from_axis,
                        const float temperature) {
    DenseTensor random_tensor;
    int64_t size = size_to_axis * size_from_axis;
    random_tensor.Resize(make_ddim({size}));
    auto* random_data = ctx.template Alloc<T>(&random_tensor);
    thrust::counting_iterator<int64_t> index_sequence_begin(0);

    // generate gumbel noise
    int device_id = ctx.GetPlace().GetDeviceId();
    auto gen_cuda = paddle::framework::GetDefaultCUDAGenerator(device_id);
    if (gen_cuda->GetIsInitPy()) {
      auto seed_offset = gen_cuda->IncrementOffset(1);
      int64_t gen_offset = size * seed_offset.second;
      thrust::transform(
          index_sequence_begin,
          index_sequence_begin + size,
          thrust::device_ptr<T>(random_data),
          UniformCUDAGenerator<T>(0.00001, 1, seed_offset.first, gen_offset));
    } else {
      const unsigned int seed = std::random_device()();
      thrust::transform(index_sequence_begin,
                        index_sequence_begin + size,
                        thrust::device_ptr<T>(random_data),
                        UniformCUDAGenerator<T>(0.00001, 1, seed));
    }

    // add gumbel noise to X
    const int thread_size = 512;
    int64_t block_size = (size + thread_size) / thread_size;
    AddGumbelNoiseCUDAKernel<T><<<block_size, thread_size, 0, ctx.stream()>>>(
        input_data, output_data, random_data, temperature, size);
  }
};

}  // namespace phi
#endif

PD_REGISTER_KERNEL(
    gumbel_softmax, GPU, ALL_LAYOUT, phi::GumbelSoftmaxKernel, float, double) {}
