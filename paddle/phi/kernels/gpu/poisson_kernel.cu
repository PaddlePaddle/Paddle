/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include <hiprand/hiprand_kernel.h>
#endif
#ifdef __HIPCC__
#include <hiprand_kernel.h>
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/poisson_kernel.h"

namespace phi {

template <typename T>
struct PoissonCudaFunctor {
 public:
  PoissonCudaFunctor(const T* in,
                     T* out,
                     unsigned int seed,
                     unsigned int offset)
      : in_(in), out_(out), seed_(seed), offset_(offset) {}

  __device__ void operator()(int64_t idx) {
#ifdef __NVCC__
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed_, idx, offset_, &state);
    out_[idx] = static_cast<T>(hiprand_poisson(&state, in_[idx]));
#elif __HIPCC__
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed_, idx, offset_, &state);
    out_[idx] = static_cast<T>(hiprand_poisson(&state, in_[idx]));
#endif
  }

 private:
  const T* in_;
  T* out_;
  const unsigned int seed_;
  const unsigned int offset_;
};

template <typename T, typename Context>
void PoissonKernel(const Context& ctx, const DenseTensor& x, DenseTensor* out) {
  const T* x_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(out);
  auto size = x.numel();

  auto gen_cuda = ctx.GetGenerator();
  auto seed_offset = gen_cuda->IncrementOffset(20);
  uint64_t seed = seed_offset.first;
  uint64_t offset = seed_offset.second;

  phi::funcs::ForRange<Context> for_range(ctx, size);

  PoissonCudaFunctor<T> functor(x_data, out_data, seed, offset);
  for_range(functor);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    poisson, GPU, ALL_LAYOUT, phi::PoissonKernel, float, double) {}
