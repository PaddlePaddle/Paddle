#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifdef __NVCC__
#include <hiprand/hiprand_kernel.h>
#endif
#ifdef __HIPCC__
#include <hiprand_kernel.h>
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/poisson_kernel.h"

namespace phi {

template <typename T>
__global__ void GetPoisson(
    const T* in, T* out, const int N, unsigned int seed, unsigned int offset) {
  CUDA_KERNEL_LOOP_TYPE(idx, N, int64_t) {
#ifdef __NVCC__
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, idx, offset, &state);
    out[idx] = static_cast<T>(hiprand_poisson(&state, in[idx]));
#elif __HIPCC__
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, idx, offset, &state);
    out[idx] = static_cast<T>(hiprand_poisson(&state, in[idx]));
#endif
  }
}

template <typename T, typename Context>
void PoissonKernel(const Context& ctx, const DenseTensor& x, DenseTensor* out) {
  const T* x_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(out);
  const int size = x.numel();
  const int kMaxBlockDim = 256;

  int block_size = std::min(kMaxBlockDim, ctx.GetMaxThreadsPerBlock());
  dim3 dim_block(block_size);
  dim3 dim_grid((size + block_size - 1) / block_size);
  phi::backends::gpu::LimitGridDim(ctx, &dim_grid);

  auto gen_cuda = ctx.GetGenerator();
  auto seed_offset = gen_cuda->IncrementOffset(20);
  uint64_t seed = seed_offset.first;
  uint64_t offset = seed_offset.second;
  GetPoisson<T><<<dim_grid, dim_block>>>(x_data, out_data, size, seed, offset);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    poisson, GPU, ALL_LAYOUT, phi::PoissonKernel, float, double) {}
