#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/prelu_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/gpu/prelu_funcs.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

namespace phi {

enum PRELU_MODE { Element, ChannelFirst, ChannelLast, PRELU_Scalar };

template <typename T>
__global__ void PReluOpGradKernel(const T* x_ptr,
                                  const T* alpha_ptr,
                                  const T* out_grad_ptr,
                                  T* x_grad_ptr,
                                  T* alpha_grad_ptr,
                                  size_t channel_num,
                                  size_t plane_size,
                                  size_t spatial_size,
                                  size_t numel,
                                  PRELU_MODE mode) {
  CUDA_KERNEL_LOOP(index, numel) {
    T scale;
    if (mode == Element) {
      size_t element_index = index % spatial_size;
      scale = alpha_ptr[element_index];
    } else if (mode == ChannelFirst) {
      size_t temp = index / plane_size;
      size_t channel_index = temp % channel_num;
      scale = alpha_ptr[channel_index];
    } else if (mode == ChannelLast) {
      size_t channel_index = index % channel_num;
      scale = alpha_ptr[channel_index];
    } else {
      scale = alpha_ptr[0];
    }
    T x = x_ptr[index];
    T out_grad = out_grad_ptr[index];
    T zero = static_cast<T>(0);
    if (x_grad_ptr != nullptr)
      x_grad_ptr[index] = (x > zero) ? out_grad : scale * out_grad;
    if (alpha_grad_ptr != nullptr)
      alpha_grad_ptr[index] = (x > zero) ? zero : x * out_grad;
  }
}

template <typename T>
class PreluOpGradFunctor {
 public:
  void operator()(gpuStream_t stream,
                  const T* x,
                  const T* alpha,
                  const T* out_grad,
                  T* x_grad,
                  T* alpha_grad,
                  const DDim& input_dims,
                  PRELU_MODE mode) {
    size_t numel = 1;
    for (size_t i = 0; i < input_dims.size(); ++i) {
      numel *= input_dims[i];
    }
    size_t plane_size = numel / input_dims[0] / input_dims[1];
    size_t spatial_size = numel / input_dims[0];
    size_t channel =
        mode == ChannelLast ? input_dims[input_dims.size() - 1] : input_dims[1];

    PReluOpGradKernel<T>
        <<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
            x,
            alpha,
            out_grad,
            x_grad,
            alpha_grad,
            channel,
            plane_size,
            spatial_size,
            numel,
            mode);
  }
};

template <typename T, typename Context>
void PReluGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& alpha,
                     const DenseTensor& out_grad,
                     const std::string& data_format,
                     const std::string& mode,
                     DenseTensor* x_grad,
                     DenseTensor* alpha_grad) {
  dev_ctx.template Alloc<T>(x_grad);

  const T* x_ptr = x.data<T>();
  const T* alpha_ptr = alpha.data<T>();
  const T* out_grad_ptr = out_grad.data<T>();
  T* x_grad_ptr = x_grad ? dev_ctx.template Alloc<T>(x_grad) : nullptr;
  T* alpha_grad_ptr =
      alpha_grad ? dev_ctx.template Alloc<T>(alpha_grad) : nullptr;

  if (!x_grad && !alpha_grad) return;

  int numel = x.numel();
  auto dim = x.dims();
  auto x_rank = dim.size();
  std::vector<int> input_shape = phi::vectorize<int>(dim);
  auto stream = dev_ctx.stream();

  T* alpha_grad_tmp_ptr;
  DenseTensor alpha_grad_tmp;
  if (alpha_grad_ptr == nullptr) {
    alpha_grad_tmp_ptr = alpha_grad_ptr;
  } else {
    DenseTensorMeta alpha_grad_meta(
        alpha_grad->dtype(), dim, alpha_grad->layout());
    alpha_grad_tmp = phi::Empty(dev_ctx, std::move(alpha_grad_meta));
    alpha_grad_tmp_ptr = alpha_grad_tmp.data<T>();
  }

  PRELU_MODE m;
  bool channel_last = false;
  if (mode == "element") {
    m = Element;
  } else if (mode == "channel") {
    channel_last = data_format == "NHWC";
    m = channel_last ? ChannelLast : ChannelFirst;
  } else {
    m = PRELU_Scalar;
  }
  PreluOpGradFunctor<T> prelu_grad;
  prelu_grad(stream,
             x_ptr,
             alpha_ptr,
             out_grad_ptr,
             x_grad_ptr,
             alpha_grad_tmp_ptr,
             dim,
             m);

  if (alpha_grad_tmp_ptr == nullptr) return;

  std::vector<int> reduce_dims;
  for (size_t i = 0; i < dim.size(); i++) {
    if (mode == "channel" && !channel_last && i == 1) continue;
    if (mode == "channel" && channel_last && i == dim.size() - 1) continue;
    if (mode == "element" && i != 0) continue;
    reduce_dims.push_back(i);
  }

  phi::funcs::ReduceKernel<T, T, kps::AddFunctor, kps::IdentityFunctor<T>>(
      static_cast<const phi::GPUContext&>(dev_ctx),
      alpha_grad_tmp,
      alpha_grad,
      kps::IdentityFunctor<T>(),
      reduce_dims);
}

}  // namespace phi

PD_REGISTER_KERNEL(prelu_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::PReluGradKernel,
                   float,
                   phi::dtype::float16,
                   double) {}
