#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/rrelu_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/gpu/prelu_funcs.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

namespace phi {

template <typename T>
__global__ void RReluOpGradKernel(const T* x_ptr,
                                  const T* noise_ptr,
                                  const T* out_grad_ptr,
                                  T* x_grad_ptr,
                                  int numel) {
  CUDA_KERNEL_LOOP(index, numel) {
    T scale = noise_ptr[index];
    T x = x_ptr[index];
    T out_grad = out_grad_ptr[index];
    T zero = static_cast<T>(0);
    x_grad_ptr[index] = (x < zero) ? scale * out_grad : out_grad;
  }
}

template <typename T>
class RReluOpGradFunctor {
 public:
  void operator()(gpuStream_t stream,
                  const T* x,
                  const T* noise,
                  const T* out_grad,
                  T* x_grad,
                  int numel) {
    RReluOpGradKernel<T>
        <<<PADDLE_GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, stream>>>(
            x, noise, out_grad, x_grad, numel);
  }
};

template <typename T, typename Context>
void RReluGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& noise,
                     const DenseTensor& out_grad,
                     DenseTensor* x_grad) {
  if (!x_grad) return;
  dev_ctx.template Alloc<T>(x_grad);

  const T* x_ptr = x.data<T>();
  const T* n_ptr = noise.data<T>();
  const T* out_grad_ptr = out_grad.data<T>();
  T* x_grad_ptr = dev_ctx.template Alloc<T>(x_grad);

  int numel = x.numel();
  auto stream = dev_ctx.stream();

  RReluOpGradFunctor<T> rrelu_grad;
  rrelu_grad(stream, x_ptr, n_ptr, out_grad_ptr, x_grad_ptr, numel);
}

}  // namespace phi

PD_REGISTER_KERNEL(rrelu_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::RReluGradKernel,
                   float,
                   phi::dtype::float16,
                   double) {}
