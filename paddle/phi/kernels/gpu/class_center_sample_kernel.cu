#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_HIP
#include <hiprand.h>
#include <hiprand_kernel.h>

#include <hipcub/hipcub.hpp>
typedef hiprandState hiprandState;
namespace cub = hipcub;
#else
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hipcub/hipcub.hpp>
#endif

#include <iterator>
#include <random>

#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/platform/enforce.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/ProcessGroup.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {
#define CUDA_KERNEL_LOOP(i, n)                            \
  for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x, \
               step = blockDim.x * gridDim.x;             \
       i < (n);                                           \
       i += step)

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

inline int32_t NumBlocks(const int32_t n) {
  return std::min((n + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T>
__global__ void RandomSampleClassCenter(const int64_t n,
                                        int64_t seed,
                                        int64_t increment,
                                        const int64_t max_val,
                                        T* buffer) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState;
  size_t local_seed =
      (static_cast<size_t>(seed) + 0x9E3779B9U +
       (static_cast<size_t>(id) << 6U) + (static_cast<size_t>(id) >> 2U));
#ifdef PADDLE_WITH_HIP
  hiprand_init(local_seed, id, increment, &localState);
  CUDA_KERNEL_LOOP(i, n) {
    buffer[i] = static_cast<T>(hiprand(&localState) % max_val);
  }
#else
  hiprand_init(local_seed, id, increment, &localState);
  CUDA_KERNEL_LOOP(i, n) {
    buffer[i] = static_cast<T>(hiprand(&localState) % max_val);
  }
#endif
}

template <typename T>
__global__ void Range(const int64_t n, T* out) {
  CUDA_KERNEL_LOOP(i, n) { out[i] = static_cast<T>(i); }
}

template <typename T>
__global__ void MarkPositiveClassCenter(const int64_t n,
                                        const int64_t rank,
                                        const T* class_interval_ptr,
                                        const int num_classes,
                                        const T* labels,
                                        T* out) {
  CUDA_KERNEL_LOOP(i, n) {
    T label = labels[i] - class_interval_ptr[rank];
    if (label >= 0 && label < num_classes) {
      out[label] = label - num_classes;
    }
  }
}

template <typename T>
__device__ void FindIntervalIndex(const T* class_interval_ptr,
                                  const int64_t nranks,
                                  const T value,
                                  int64_t* find_index) {
  int64_t start = 0;
  int64_t end = nranks;
  int64_t mid = ((end - start) >> 1) + start + 1;
  while (start < end) {
    if (class_interval_ptr[mid] == value) break;
    if (class_interval_ptr[mid] > value)
      end = mid - 1;
    else
      start = mid;
    mid = ((end - start) >> 1) + start + 1;
  }
  *find_index = min(mid, end);
}

template <typename T>
__global__ void GetClassCenterBound(const int64_t n,
                                    const int64_t nranks,
                                    const T* class_interval_ptr,
                                    const T* key_ptr,
                                    const T* value_ptr,
                                    T* bound_index,
                                    T* bound_value) {
  CUDA_KERNEL_LOOP(i, n) {
    if (i != 0) {
      int64_t cur_index, pre_index;
      FindIntervalIndex(class_interval_ptr, nranks, key_ptr[i], &cur_index);
      FindIntervalIndex(class_interval_ptr, nranks, key_ptr[i - 1], &pre_index);
      if (cur_index > pre_index) {
        assert(cur_index < nranks);
#pragma unroll
        for (int32_t j = pre_index + 1; j <= cur_index; ++j) {
          bound_index[j] = static_cast<T>(i);
          bound_value[j] = value_ptr[i];
        }
      }
    }
  }
  CUDA_KERNEL_LOOP(i, nranks + 1) {
    int64_t first_index, last_index;
    FindIntervalIndex(class_interval_ptr, nranks, key_ptr[0], &first_index);
    FindIntervalIndex(class_interval_ptr, nranks, key_ptr[n - 1], &last_index);
    if (i <= first_index) {
      bound_index[i] = 0;
      bound_value[i] = value_ptr[0];
    } else if (i > last_index) {
      bound_index[i] = n;
      bound_value[i] = value_ptr[n - 1] + 1;
    }
  }
}

template <typename T>
__global__ void GetRemappedLabel(const int64_t n,
                                 const int64_t nranks,
                                 const T* sampled_class_interval_ptr,
                                 const T* bound_index,
                                 const T* bound_value,
                                 const T* label_map_key,
                                 T* label_map_value,
                                 T* mapped_label) {
  CUDA_KERNEL_LOOP(i, n) {
#pragma unroll
    for (int64_t j = 0; j < nranks; j++) {
      if (i >= bound_index[j] && i < bound_index[j + 1]) {
        label_map_value[i] =
            label_map_value[i] - bound_value[j] + sampled_class_interval_ptr[j];
      }
    }
    mapped_label[label_map_key[i]] = label_map_value[i];
  }
}

// aligned vector generates vectorized load/store on CUDA
template <typename T, int Size>
struct alignas(sizeof(T) * Size) AlignedVector {
  T val[Size];
};

template <typename T>
inline int VectorizedSize(const T* pointer) {
  uint64_t address = reinterpret_cast<uint64_t>(pointer);
  constexpr int vec4 = std::alignment_of<AlignedVector<T, 4>>::value;  // NOLINT
  if (address % vec4 == 0) {
    return 4;
  }
  return 1;
}

#undef CUDA_KERNEL_LOOP

template <typename T>
class NotEqualToPreviousAdjacentIterator {
 public:
  using self_type = NotEqualToPreviousAdjacentIterator;
  using value_type = T;
  using difference_type = std::ptrdiff_t;
  using pointer = T*;
  using reference = T;
  using iterator_category = std::input_iterator_tag;

 public:
  __host__ __device__ __forceinline__
  NotEqualToPreviousAdjacentIterator(const T* arr, int64_t offset)
      : arr_(arr), offset_(offset) {}

  __host__ __device__ __forceinline__ reference operator*() const {
    return offset_ == 0 ? 0 : (arr_[offset_] == arr_[offset_ - 1] ? 0 : 1);
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator+(Distance n) const {
    self_type ret(arr_, offset_ + n);
    return ret;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ self_type operator-(Distance n) const {
    self_type ret(arr_, offset_ - n);
    return ret;
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ reference operator[](Distance n) const {
    return *(*this + n);
  }

 private:
  const T* arr_;
  int64_t offset_;
};

template <typename T>
struct ActualNumSampledFunctor {
  __host__ __device__ __forceinline__ T operator()(const T& a,
                                                   const T& b) const {
    return max(num_samples, (b - a));
  }
  T num_samples;
  explicit ActualNumSampledFunctor(const T num) : num_samples(num) {}
};

template <typename T, typename Context>
class MemoryBuffer {
 public:
  MemoryBuffer(const int num_buffer_ele,
               const int num_temp_ele,
               const int nranks,
               const Context& dev_ctx) {
    offset1 = 0;
    offset2 = offset1 + num_buffer_ele;
    offset3 = offset2 + num_buffer_ele;
    offset4 = offset3 + num_buffer_ele;
    offset5 = offset4 + num_buffer_ele;
    offset6 = offset5 + (nranks + 1);
    offset7 = offset6 + (nranks + 1);
    offset8 = offset7 + (nranks + 1);
    offset9 = offset8 + num_temp_ele;

    buffer.Resize({4 * num_buffer_ele + 3 * (nranks + 1) + num_temp_ele});
    buffer_ptr = dev_ctx.template Alloc<T>(&buffer);
  }

  T* cub_sort_keys_ptr() { return buffer_ptr + offset1; }
  T* cub_sort_keys_out_ptr() { return buffer_ptr + offset2; }
  T* cub_sort_values_ptr() { return buffer_ptr + offset3; }
  T* cub_sort_values_out_ptr() { return buffer_ptr + offset4; }
  T* bound_index_ptr() { return buffer_ptr + offset5; }
  T* bound_value_ptr() { return buffer_ptr + offset6; }
  T* class_interval_ptr() { return buffer_ptr + offset7; }
  void* cub_temp_storage_ptr() {
    return reinterpret_cast<void*>(buffer_ptr + offset8);
  }

 private:
  DenseTensor buffer;
  T* buffer_ptr;
  int offset1;
  int offset2;
  int offset3;
  int offset4;
  int offset5;
  int offset6;
  int offset7;
  int offset8;
  int offset9;
};

template <typename T, typename Context>
void ClassCenterSampleKernel(const Context& dev_ctx,
                             const DenseTensor& label,
                             int num_classes,
                             int num_samples,
                             int ring_id,
                             int rank,
                             int nranks,
                             bool fix_seed,
                             int seed,
                             DenseTensor* remapped_label,
                             DenseTensor* sampled_local_class_center) {
  PADDLE_ENFORCE_GT(num_classes,
                    0,
                    errors::InvalidArgument(
                        "The value 'num_classes' for Op(class_center_sample) "
                        "must be greater than 0, "
                        "but the value given is %d.",
                        num_classes));

  PADDLE_ENFORCE_GT(num_samples,
                    0,
                    errors::InvalidArgument(
                        "The value 'num_samples' for Op(class_center_sample) "
                        "must be greater than 0, "
                        "but the value given is %d.",
                        num_samples));

  PADDLE_ENFORCE_LE(num_samples,
                    num_classes,
                    errors::InvalidArgument(
                        "The value 'num_samples' for Op(class_center_sample) "
                        "must be less than or equal to %d, "
                        "but the value given is %d.",
                        num_classes,
                        num_samples));

  auto place = dev_ctx.GetPlace();

  int batch_size = label.numel();
  // Algorithm:
  // We first randomly generate a value in [0, num_classes) on each position
  // in a array(shape[num_classes]). Then, we mark the element as negative
  // value in the array according input label. Now, we can sort the array
  // by ascending to ensure that the positive class center always in the
  // front of the sorted array. So, we can get the sampled class center
  // index by sorted keys. Finally, we can get the rempped label by remap
  // the input label according sampled class center.

  // step 1: Calculate num classes per device using nccl all reduce
  std::vector<T> shard_dim_vec(nranks + 1, 0);
  shard_dim_vec[rank + 1] = num_classes;
  DenseTensor num_classes_per_device;
  paddle::framework::TensorFromVector(
      shard_dim_vec, dev_ctx, &num_classes_per_device);
  T* num_classes_per_device_ptr = num_classes_per_device.data<T>();

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();
    if (map->has(ring_id)) {
      // Use ProcessGroup
      paddle::distributed::ProcessGroup* pg = map->get(ring_id);
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(num_classes_per_device);
      out_tensor.push_back(num_classes_per_device);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::SUM;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      const auto& comm = paddle::platform::NCCLCommContext::Instance().Get(
          ring_id, dev_ctx.GetPlace());
      // use global calculate stream
      const auto calcu_stream =
          static_cast<GPUContext*>(
              paddle::platform::DeviceContextPool::Instance().Get(
                  dev_ctx.GetPlace()))
              ->stream();
      PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::ncclAllReduce(
          num_classes_per_device_ptr,
          num_classes_per_device_ptr,
          num_classes_per_device.numel(),
          paddle::platform::ToNCCLDataType(
              paddle::framework::TransToProtoVarType(
                  num_classes_per_device.dtype())),
          ncclSum,
          comm->comm(),
          calcu_stream));
    }
  }
#endif

  // step 2: Determine temporary device storage requirements
  int num_buffer_ele = std::max(batch_size, num_classes);
  size_t cub_sort_temp_store_size = 0;
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceRadixSort::SortPairs<T, T>(nullptr,
                                             cub_sort_temp_store_size,
                                             nullptr,
                                             nullptr,
                                             nullptr,
                                             nullptr,
                                             num_buffer_ele,
                                             0,
                                             sizeof(T) * 8,
                                             dev_ctx.stream())));

  size_t cub_sum_temp_store_size = 0;
  NotEqualToPreviousAdjacentIterator<T> unique_counting_iter_temp(nullptr, 0);
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceScan::InclusiveSum<NotEqualToPreviousAdjacentIterator<T>, T*>(
          nullptr,
          cub_sum_temp_store_size,
          unique_counting_iter_temp,
          nullptr,
          batch_size,
          dev_ctx.stream())));

  size_t cub_scan_temp_store_size = 0;
  ActualNumSampledFunctor<T> actual_num_sampled_op_temp(num_samples);
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceScan::InclusiveScan(nullptr,
                                      cub_scan_temp_store_size,
                                      num_classes_per_device_ptr,
                                      num_classes_per_device_ptr,
                                      actual_num_sampled_op_temp,
                                      nranks + 1,
                                      dev_ctx.stream())));

  size_t cub_temp_storage_bytes =
      std::max(std::max(cub_sort_temp_store_size, cub_scan_temp_store_size),
               cub_sum_temp_store_size);
  int num_temp_ele = cub_temp_storage_bytes / sizeof(T) + 1;

  // step 3: Alloc buffer memory so that we can reuse allocated memory
  MemoryBuffer<T, Context> memory_buffer =
      MemoryBuffer<T, Context>(num_buffer_ele, num_temp_ele, nranks, dev_ctx);

  T* cub_sort_keys_ptr = memory_buffer.cub_sort_keys_ptr();
  T* cub_sort_keys_out_ptr = memory_buffer.cub_sort_keys_out_ptr();
  T* cub_sort_values_ptr = memory_buffer.cub_sort_values_ptr();
  T* cub_sort_values_out_ptr = memory_buffer.cub_sort_values_out_ptr();
  T* bound_index_ptr = memory_buffer.bound_index_ptr();
  T* bound_value_ptr = memory_buffer.bound_value_ptr();
  T* class_interval_ptr = memory_buffer.class_interval_ptr();
  void* cub_temp_storage_ptr = memory_buffer.cub_temp_storage_ptr();

  // step 4: Calculate class interval among nranks
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceScan::InclusiveSum(cub_temp_storage_ptr,
                                     cub_temp_storage_bytes,
                                     num_classes_per_device_ptr,
                                     class_interval_ptr,
                                     nranks + 1,
                                     dev_ctx.stream())));

  // step 5: random sample negative class center
  uint64_t seed_data;
  uint64_t increment;
  int vec_size = VectorizedSize<T>(cub_sort_keys_ptr);
  auto offset = ((num_classes - 1) /
                     (NumBlocks(num_classes) * kNumCUDAThreads * vec_size) +
                 1) *
                vec_size;
  // auto gen_cuda = paddle::framework::DefaultCUDAGenerator(device_id);
  auto gen_cuda = dev_ctx.GetGenerator();
  if (!fix_seed) {
    auto seed_offset = gen_cuda->IncrementOffset(offset);
    seed_data = seed_offset.first;
    increment = seed_offset.second;
  } else {
    seed_data = seed + rank;
    increment = offset;
  }
  RandomSampleClassCenter<T>
      <<<NumBlocks(num_classes), kNumCUDAThreads, 0, dev_ctx.stream()>>>(
          num_classes, seed_data, increment, num_classes, cub_sort_keys_ptr);

  // step 6: mark positive class center as negative value
  // fill the sort values to index 0, 1, ..., batch_size-1
  MarkPositiveClassCenter<T>
      <<<NumBlocks(batch_size), kNumCUDAThreads, 0, dev_ctx.stream()>>>(
          batch_size,
          rank,
          class_interval_ptr,
          num_classes,
          label.data<T>(),
          cub_sort_keys_ptr);
  Range<T><<<NumBlocks(num_buffer_ele), kNumCUDAThreads, 0, dev_ctx.stream()>>>(
      num_buffer_ele, cub_sort_values_ptr);

  // step 7: sort class center by ascending, so that positive class center
  // always be sampled.
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceRadixSort::SortPairs<T, T>(cub_temp_storage_ptr,
                                             cub_temp_storage_bytes,
                                             cub_sort_keys_ptr,
                                             cub_sort_keys_out_ptr,
                                             cub_sort_values_ptr,
                                             cub_sort_values_out_ptr,
                                             num_classes,
                                             0,
                                             sizeof(T) * 8,
                                             dev_ctx.stream())));

  // step 8: sort input label ascending
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceRadixSort::SortPairs<T, T>(cub_temp_storage_ptr,
                                             cub_temp_storage_bytes,
                                             label.data<T>(),
                                             cub_sort_keys_out_ptr,
                                             cub_sort_values_ptr,
                                             cub_sort_keys_ptr,
                                             batch_size,
                                             0,
                                             sizeof(T) * 8,
                                             dev_ctx.stream())));

  // step 9: Calculate new index using InclusiveSum on ascending sorted input
  // label
  NotEqualToPreviousAdjacentIterator<T> unique_counting_iter(
      cub_sort_keys_out_ptr, 0);
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceScan::InclusiveSum<NotEqualToPreviousAdjacentIterator<T>, T*>(
          cub_temp_storage_ptr,
          cub_temp_storage_bytes,
          unique_counting_iter,
          cub_sort_values_ptr,
          batch_size,
          dev_ctx.stream())));

  // step 10: Calculate new class center bound among ranks
  GetClassCenterBound<T>
      <<<NumBlocks(batch_size), kNumCUDAThreads, 0, dev_ctx.stream()>>>(
          batch_size,
          nranks,
          class_interval_ptr,
          cub_sort_keys_out_ptr,
          cub_sort_values_ptr,
          bound_index_ptr,
          bound_value_ptr);

  // step 11: Calculate actual number of sampled class per device.
  // Since maybe num_positive_class_center > num_samples,
  // we need to ensure all positive class center per device are sampled.
  ActualNumSampledFunctor<T> actual_num_sampled_op(num_samples);
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceScan::InclusiveScan(cub_temp_storage_ptr,
                                      cub_temp_storage_bytes,
                                      bound_value_ptr,
                                      num_classes_per_device_ptr,
                                      actual_num_sampled_op,
                                      nranks + 1,
                                      dev_ctx.stream())));

  // step 12: Calculate actual sampled class interval among nranks
  PADDLE_ENFORCE_GPU_SUCCESS(
      (hipcub::DeviceScan::InclusiveSum(cub_temp_storage_ptr,
                                     cub_temp_storage_bytes,
                                     num_classes_per_device_ptr,
                                     class_interval_ptr,
                                     nranks + 1,
                                     dev_ctx.stream())));

  // step 13: Get remapped label for output
  GetRemappedLabel<T>
      <<<NumBlocks(batch_size), kNumCUDAThreads, 0, dev_ctx.stream()>>>(
          batch_size,
          nranks,
          class_interval_ptr,
          bound_index_ptr,
          bound_value_ptr,
          cub_sort_keys_ptr,
          cub_sort_values_ptr,
          dev_ctx.template Alloc<T>(remapped_label));

  // step 14: Get sampled class center for output
  phi::Copy<Context>(dev_ctx,
                     num_classes_per_device,
                     phi::CPUPlace(),
                     true,
                     &num_classes_per_device);
  T actual_num_samples = num_classes_per_device.data<T>()[rank + 1];
  sampled_local_class_center->Resize(phi::make_ddim({actual_num_samples}));

  T* sampled_local_class_center_ptr =
      dev_ctx.template Alloc<T>(sampled_local_class_center);
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       sampled_local_class_center_ptr,
                       dev_ctx.GetPlace(),
                       cub_sort_values_out_ptr,
                       actual_num_samples * sizeof(T),
                       nullptr);
}
}  // namespace phi

PD_REGISTER_KERNEL(class_center_sample,
                   GPU,
                   ALL_LAYOUT,
                   phi::ClassCenterSampleKernel,
                   int64_t,
                   int) {}
