#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/pad3d_kernel.h"

#include <algorithm>

#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/complex.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename T>
__global__ void Pad3DConstNCDHW(const int nthreads,
                                const T* in_data,
                                const int num,
                                const int channels,
                                const int in_depth,
                                const int in_height,
                                const int in_width,
                                const int out_depth,
                                const int out_height,
                                const int out_width,
                                const int pad_front,
                                const int pad_top,
                                const int pad_left,
                                T value,
                                T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;
    out_data[index] =
        (in_d < 0 || in_h < 0 || in_w < 0 || in_d >= in_depth ||
         in_h >= in_height || in_w >= in_width)
            ? value
            : in_data[nc * in_depth * in_height * in_width +
                      in_d * in_height * in_width + in_h * in_width + in_w];
  }
}

template <typename T>
__global__ void Pad3DConstNDHWC(const int nthreads,
                                const T* in_data,
                                const int num,
                                const int channels,
                                const int in_depth,
                                const int in_height,
                                const int in_width,
                                const int out_depth,
                                const int out_height,
                                const int out_width,
                                const int pad_front,
                                const int pad_top,
                                const int pad_left,
                                T value,
                                T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;
    const int in_d = out_d - pad_front;
    const int in_h = out_h - pad_top;
    const int in_w = out_w - pad_left;

    out_data[index] =
        (in_d < 0 || in_h < 0 || in_w < 0 || in_d >= in_depth ||
         in_h >= in_height || in_w >= in_width)
            ? value
            : in_data[n * in_depth * in_height * in_width * channels +
                      in_d * in_height * in_width * channels +
                      in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DReflectNCDHW(const int nthreads,
                                  const T* in_data,
                                  const int num,
                                  const int channels,
                                  const int in_depth,
                                  const int in_height,
                                  const int in_width,
                                  const int out_depth,
                                  const int out_height,
                                  const int out_width,
                                  const int pad_front,
                                  const int pad_top,
                                  const int pad_left,
                                  T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);                     // reflect by 0
    in_d = min(in_d, 2 * in_depth - in_d - 2);   // reflect by in_depth
    in_h = max(in_h, -in_h);                     // reflect by 0
    in_h = min(in_h, 2 * in_height - in_h - 2);  // reflect by in_height
    in_w = max(in_w, -in_w);                     // reflect by 0
    in_w = min(in_w, 2 * in_width - in_w - 2);   // reflect by in_width
    out_data[index] =
        in_data[(nc * in_depth * in_height + in_d * in_height + in_h) *
                    in_width +
                in_w];
  }
}

template <typename T>
__global__ void Pad3DReflectNDHWC(const int nthreads,
                                  const T* in_data,
                                  const int num,
                                  const int channels,
                                  const int in_depth,
                                  const int in_height,
                                  const int in_width,
                                  const int out_depth,
                                  const int out_height,
                                  const int out_width,
                                  const int pad_front,
                                  const int pad_top,
                                  const int pad_left,
                                  T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;
    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);
    in_d = min(in_d, 2 * in_depth - in_d - 2);
    in_h = max(in_h, -in_h);
    in_h = min(in_h, 2 * in_height - in_h - 2);
    in_w = max(in_w, -in_w);
    in_w = min(in_w, 2 * in_width - in_w - 2);

    out_data[index] = in_data[n * in_depth * in_height * in_width * channels +
                              in_d * in_height * in_width * channels +
                              in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DReplicateNCDHW(const int nthreads,
                                    const T* in_data,
                                    const int num,
                                    const int channels,
                                    const int in_depth,
                                    const int in_height,
                                    const int in_width,
                                    const int out_depth,
                                    const int out_height,
                                    const int out_width,
                                    const int pad_front,
                                    const int pad_top,
                                    const int pad_left,
                                    T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    out_data[index] =
        in_data[(nc * in_depth * in_height + in_d * in_height + in_h) *
                    in_width +
                in_w];
  }
}

template <typename T>
__global__ void Pad3DReplicateNDHWC(const int nthreads,
                                    const T* in_data,
                                    const int num,
                                    const int channels,
                                    const int in_depth,
                                    const int in_height,
                                    const int in_width,
                                    const int out_depth,
                                    const int out_height,
                                    const int out_width,
                                    const int pad_front,
                                    const int pad_top,
                                    const int pad_left,
                                    T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    out_data[index] = in_data[n * in_depth * in_height * in_width * channels +
                              in_d * in_height * in_width * channels +
                              in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DCircularNCDHW(const int nthreads,
                                   const T* in_data,
                                   const int num,
                                   const int channels,
                                   const int in_depth,
                                   const int in_height,
                                   const int in_width,
                                   const int out_depth,
                                   const int out_height,
                                   const int out_width,
                                   const int pad_front,
                                   const int pad_top,
                                   const int pad_left,
                                   T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int nc = index / out_width;

    const int out_w = index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    out_data[index] =
        in_data[(nc * in_depth * in_height + in_d * in_height + in_h) *
                    in_width +
                in_w];
  }
}

template <typename T>
__global__ void Pad3DCircularNDHWC(const int nthreads,
                                   const T* in_data,
                                   const int num,
                                   const int channels,
                                   const int in_depth,
                                   const int in_height,
                                   const int in_width,
                                   const int out_depth,
                                   const int out_height,
                                   const int out_width,
                                   const int pad_front,
                                   const int pad_top,
                                   const int pad_left,
                                   T* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index / channels;
    const int c = index % channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    out_data[index] = in_data[n * in_depth * in_height * in_width * channels +
                              in_d * in_height * in_width * channels +
                              in_h * in_width * channels + in_w * channels + c];
  }
}

template <typename T, typename Context>
void Pad3dKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const IntArray& paddings,
                 const std::string& mode,
                 float pad_value,
                 const std::string& data_format,
                 DenseTensor* out) {
  std::vector<int64_t> pads = paddings.GetData();

  auto in_dims = x.dims();
  const T* in_data = x.data<T>();
  auto out_dims = out->dims();
  T value = static_cast<T>(pad_value);

  if (data_format == "NCDHW") {
    out_dims[0] = in_dims[0];
    out_dims[1] = in_dims[1];
    out_dims[2] = in_dims[2] + pads[4] + pads[5];
    out_dims[3] = in_dims[3] + pads[2] + pads[3];
    out_dims[4] = in_dims[4] + pads[0] + pads[1];
  } else {
    out_dims[0] = in_dims[0];
    out_dims[1] = in_dims[1] + pads[4] + pads[5];
    out_dims[2] = in_dims[2] + pads[2] + pads[3];
    out_dims[3] = in_dims[3] + pads[0] + pads[1];
    out_dims[4] = in_dims[4];
  }
  out->Resize(out_dims);
  T* out_data = dev_ctx.template Alloc<T>(out);

  int channels = in_dims[1];
  int in_depth = in_dims[2];
  int in_height = in_dims[3];
  int in_width = in_dims[4];
  int out_depth = out_dims[2];
  int out_height = out_dims[3];
  int out_width = out_dims[4];
  if (data_format == "NDHWC") {
    channels = in_dims[4];
    in_depth = in_dims[1];
    in_height = in_dims[2];
    in_width = in_dims[3];
    out_depth = out_dims[1];
    out_height = out_dims[2];
    out_width = out_dims[3];
  }

  if (mode == "reflect") {
    PADDLE_ENFORCE_GT(
        in_depth,
        pads[4],
        errors::InvalidArgument("The depth of Input(X)'s dimension should be "
                                "greater than pad_front"
                                " in reflect mode"
                                ", but received depth(%d) and pad_front(%d).",
                                in_depth,
                                pads[4]));
    PADDLE_ENFORCE_GT(
        in_depth,
        pads[5],
        errors::InvalidArgument("The depth of Input(X)'s dimension should be "
                                "greater than pad_back"
                                " in reflect mode"
                                ", but received depth(%d) and pad_back(%d).",
                                in_depth,
                                pads[5]));

    PADDLE_ENFORCE_GT(
        in_height,
        pads[2],
        errors::InvalidArgument("The height of Input(X)'s dimension should be "
                                "greater than pad_top"
                                " in reflect mode"
                                ", but received depth(%d) and pad_top(%d).",
                                in_height,
                                pads[2]));
    PADDLE_ENFORCE_GT(
        in_height,
        pads[3],
        errors::InvalidArgument("The height of Input(X)'s dimension should be "
                                "greater than pad_bottom"
                                " in reflect mode"
                                ", but received depth(%d) and pad_bottom(%d).",
                                in_height,
                                pads[3]));

    PADDLE_ENFORCE_GT(
        in_width,
        pads[0],
        errors::InvalidArgument("The width of Input(X)'s dimension should be "
                                "greater than pad_left"
                                " in reflect mode"
                                ", but received depth(%d) and pad_left(%d).",
                                in_width,
                                pads[0]));
    PADDLE_ENFORCE_GT(
        in_width,
        pads[1],
        errors::InvalidArgument("The width of Input(X)'s dimension should be "
                                "greater than pad_right"
                                " in reflect mode"
                                ", but received depth(%d) and pad_right(%d).",
                                in_width,
                                pads[1]));
  } else if (mode == "circular" || mode == "replicate") {
    PADDLE_ENFORCE_NE(in_depth * in_height * in_width,
                      0,
                      errors::InvalidArgument(
                          "The input tensor size can not be 0 for circular "
                          "or replicate padding mode."));
  }

  const int pad_left = pads[0];
  const int pad_top = pads[2];
  const int pad_front = pads[4];
  const int num = in_dims[0];

  auto stream = dev_ctx.stream();
  int block = PADDLE_CUDA_NUM_THREADS;
  const int out_size = out->numel();
  int grid = (out_size + block - 1) / block;

  if (data_format == "NCDHW") {
    if (mode == "reflect") {
      Pad3DReflectNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                       in_data,
                                                       num,
                                                       channels,
                                                       in_depth,
                                                       in_height,
                                                       in_width,
                                                       out_depth,
                                                       out_height,
                                                       out_width,
                                                       pad_front,
                                                       pad_top,
                                                       pad_left,
                                                       out_data);
    } else if (mode == "replicate") {
      Pad3DReplicateNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                         in_data,
                                                         num,
                                                         channels,
                                                         in_depth,
                                                         in_height,
                                                         in_width,
                                                         out_depth,
                                                         out_height,
                                                         out_width,
                                                         pad_front,
                                                         pad_top,
                                                         pad_left,
                                                         out_data);
    } else if (mode == "circular") {
      Pad3DCircularNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                        in_data,
                                                        num,
                                                        channels,
                                                        in_depth,
                                                        in_height,
                                                        in_width,
                                                        out_depth,
                                                        out_height,
                                                        out_width,
                                                        pad_front,
                                                        pad_top,
                                                        pad_left,
                                                        out_data);
    } else {
      Pad3DConstNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                     in_data,
                                                     num,
                                                     channels,
                                                     in_depth,
                                                     in_height,
                                                     in_width,
                                                     out_depth,
                                                     out_height,
                                                     out_width,
                                                     pad_front,
                                                     pad_top,
                                                     pad_left,
                                                     value,
                                                     out_data);
    }
  } else {
    if (mode == "reflect") {
      Pad3DReflectNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                       in_data,
                                                       num,
                                                       channels,
                                                       in_depth,
                                                       in_height,
                                                       in_width,
                                                       out_depth,
                                                       out_height,
                                                       out_width,
                                                       pad_front,
                                                       pad_top,
                                                       pad_left,
                                                       out_data);
    } else if (mode == "replicate") {
      Pad3DReplicateNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                         in_data,
                                                         num,
                                                         channels,
                                                         in_depth,
                                                         in_height,
                                                         in_width,
                                                         out_depth,
                                                         out_height,
                                                         out_width,
                                                         pad_front,
                                                         pad_top,
                                                         pad_left,
                                                         out_data);
    } else if (mode == "circular") {
      Pad3DCircularNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                        in_data,
                                                        num,
                                                        channels,
                                                        in_depth,
                                                        in_height,
                                                        in_width,
                                                        out_depth,
                                                        out_height,
                                                        out_width,
                                                        pad_front,
                                                        pad_top,
                                                        pad_left,
                                                        out_data);
    } else {
      Pad3DConstNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                     in_data,
                                                     num,
                                                     channels,
                                                     in_depth,
                                                     in_height,
                                                     in_width,
                                                     out_depth,
                                                     out_height,
                                                     out_width,
                                                     pad_front,
                                                     pad_top,
                                                     pad_left,
                                                     value,
                                                     out_data);
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(pad3d,
                   GPU,
                   ALL_LAYOUT,
                   phi::Pad3dKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
