#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/distributed/collective/process_group.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/platform/collective_helper.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/cross_entropy.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/math.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/softmax.h"
#include "paddle/phi/kernels/funcs/softmax_impl.h"
#include "paddle/phi/kernels/reduce_sum_kernel.h"
#include "paddle/utils/string/string_helper.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/phi/core/distributed/nccl_comm_context.h"
#endif

namespace phi {

template <typename Context, typename T>
struct CSoftmaxWithCrossEntropyFunctor {
  void operator()(const Context& dev_ctx,
                  const DenseTensor& logits,
                  const DenseTensor& label,
                  int64_t ignore_index,
                  int ring_id,
                  int rank,
                  int nranks,
                  DenseTensor* softmax,
                  DenseTensor* loss);
};

template <typename Context, typename T>
struct CSoftmaxWithCrossEntropyProcessGroupFunctor {
  void operator()(const Context& dev_ctx,
                  const DenseTensor& logits,
                  const DenseTensor& label,
                  int64_t ignore_index,
                  int ring_id,
                  int rank,
                  int nranks,
                  DenseTensor* softmax,
                  DenseTensor* loss);
};

static constexpr int kNumCUDAThreads = 512;
static constexpr int64_t kNumMaxinumNumBlocks = 4096;

static inline int64_t NumBlocks(const int64_t N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename IndexT>
__global__ void MaskLabelByIndex(T* predicted_logits,
                                 const T* logit,
                                 const IndexT* label,
                                 const IndexT ignore_index,
                                 const int64_t start_index,
                                 const int64_t end_index,
                                 const int64_t N,
                                 const int64_t D,
                                 const int nranks) {
  CUDA_KERNEL_LOOP_TYPE(i, N, int64_t) {
    auto real_label = label[i];
    PADDLE_ENFORCE(((real_label < D * nranks) && (real_label >= 0)) ||
                       (real_label == ignore_index),
                   "The index is out of bounds, "
                   "please check whether the value of label and "
                   "input meet the class number. It should "
                   "be less than [%ld] or equal to [%ld], but received [%ld]",
                   static_cast<int64_t>(D * nranks),
                   static_cast<int64_t>(ignore_index),
                   static_cast<int64_t>(real_label));

    if (real_label >= start_index && real_label < end_index) {
      predicted_logits[i] = logit[i * D + real_label - start_index];
    }
  }
}

template <typename T, typename IndexT>
__global__ void CaculateLoss(T* loss,
                             const T* predict_logits,
                             const T* sum_exp_logits,
                             const IndexT* label,
                             const int64_t ignore_index,
                             const int64_t N) {
  CUDA_KERNEL_LOOP_TYPE(i, N, int64_t) {
    auto real_label = static_cast<int64_t>(label[i]);
    loss[i] = ignore_index == real_label
                  ? static_cast<T>(0)
                  : phi::funcs::TolerableValue<T>()(
                        phi::funcs::TolerableValue<T>()(
                            phi::funcs::real_log(sum_exp_logits[i])) -
                        predict_logits[i]);
  }
}

template <typename T, typename Context>
void CSoftmaxWithCrossEntropyKernel(const Context& dev_ctx,
                                    const DenseTensor& logits,
                                    const DenseTensor& label,
                                    int64_t ignore_index,
                                    int ring_id,
                                    int rank,
                                    int nranks,
                                    DenseTensor* softmax,
                                    DenseTensor* loss) {
  auto map = distributed::ProcessGroupMapFromGid::getInstance();
  if (map->has(ring_id)) {
    CSoftmaxWithCrossEntropyProcessGroupFunctor<phi::GPUContext, T> functor_;
    functor_(dev_ctx,
             logits,
             label,
             ignore_index,
             ring_id,
             rank,
             nranks,
             softmax,
             loss);
  } else {
    CSoftmaxWithCrossEntropyFunctor<phi::GPUContext, T> functor_;
    functor_(dev_ctx,
             logits,
             label,
             ignore_index,
             ring_id,
             rank,
             nranks,
             softmax,
             loss);
  }
}

template <typename T>
struct CSoftmaxWithCrossEntropyFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& dev_ctx,
                  const DenseTensor& logits_in,
                  const DenseTensor& label_in,
                  int64_t ignore_index,
                  int ring_id,
                  int rank,
                  int nranks,
                  DenseTensor* softmax,
                  DenseTensor* loss) {
#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
    const phi::DenseTensor* logits = &logits_in;
    const phi::DenseTensor* labels = &label_in;
    const int rid = ring_id;

    gpuStream_t stream = nullptr;
    phi::distributed::NCCLCommContext* comm_ctx = nullptr;

    comm_ctx = static_cast<phi::distributed::NCCLCommContext*>(
        dev_ctx.GetCommContext());
    PADDLE_ENFORCE_NE(comm_ctx,
                      nullptr,
                      common::errors::Unavailable(
                          "NCCLCommContext is nullptr, collective op should "
                          "has ring_id attr."));

    stream = dev_ctx.stream();
    VLOG(3) << "new comm_context_manager has ring_id " << rid;

    // allocate memory on device.
    dev_ctx.template Alloc<T>(softmax);
    dev_ctx.template Alloc<T>(loss);

    const auto& logits_dims = logits->dims();
    const auto& labels_dims = labels->dims();

    const int axis = logits_dims.size() - 1;
    const int64_t N = phi::funcs::SizeToAxis<int64_t>(axis, logits_dims);
    const int64_t D = phi::funcs::SizeFromAxis<int64_t>(axis, logits_dims);

    phi::DenseTensor logits_2d, softmax_2d, loss_2d;
    logits_2d.ShareDataWith(*logits).Resize({N, D});
    softmax_2d.ShareDataWith(*softmax).Resize({N, D});
    loss_2d.ShareDataWith(*loss).Resize({N, 1});

    auto eigen_logits = phi::funcs::EigenMatrix<T>::From(logits_2d);
    auto eigen_softmax = phi::funcs::EigenMatrix<T>::From(softmax_2d);

    // step 1, obtain logit_max
    phi::DenseTensor logits_max;
    logits_max.Resize({N, 1});
    dev_ctx.template Alloc<T>(&logits_max);

    auto eigen_logits_max = phi::funcs::EigenMatrix<T>::From(logits_max);
    Eigen::DSizes<int, 1> along_axis(1);
    eigen_logits_max.device(*dev_ctx.eigen_device()) =
        eigen_logits.maximum(along_axis);

    comm_ctx->AllReduce(&logits_max, logits_max, ncclMax, stream);

    // step 2, obtain logit - logit_max
    Eigen::DSizes<int, 2> batch_by_one(N, 1);
    Eigen::DSizes<int, 2> one_by_class(1, D);

    eigen_softmax.device(*dev_ctx.eigen_device()) =
        (eigen_logits -
         eigen_logits_max.reshape(batch_by_one).broadcast(one_by_class));

    // step 3, obtain predict target
    phi::DenseTensor predicted_logits;
    predicted_logits.Resize({N, 1});
    dev_ctx.template Alloc<T>(&predicted_logits);

    auto t = phi::EigenVector<T>::Flatten(predicted_logits);
    t.device(*dev_ctx.eigen_device()) = t.constant(static_cast<T>(0));

    const int64_t start_index = rank * D;
    const int64_t end_index = start_index + D;

    int64_t blocks = NumBlocks(N);
    int threads = kNumCUDAThreads;
    const auto& label_type = labels->dtype();

    if (label_type == phi::DataType::INT32) {
      MaskLabelByIndex<T, int32_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
          predicted_logits.data<T>(),
          softmax_2d.data<T>(),
          labels->data<int32_t>(),
          static_cast<int32_t>(ignore_index),
          start_index,
          end_index,
          N,
          D,
          nranks);
    } else if (label_type == phi::DataType::INT64) {
      MaskLabelByIndex<T, int64_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(predicted_logits.data<T>(),
                                                     softmax_2d.data<T>(),
                                                     labels->data<int64_t>(),
                                                     ignore_index,
                                                     start_index,
                                                     end_index,
                                                     N,
                                                     D,
                                                     nranks);
    }

    dev_ctx.template Alloc<T>(&predicted_logits);
    comm_ctx->AllReduce(&predicted_logits, predicted_logits, ncclSum, stream);

    // step 4, obtain exp(logit)
    eigen_softmax.device(*dev_ctx.eigen_device()) = eigen_softmax.exp();

    // step 5, obtain sum_exp_logits
    phi::DenseTensor sum_exp_logits;
    sum_exp_logits.Resize({N, 1});
    dev_ctx.template Alloc<T>(&sum_exp_logits);

    phi::SumKernel<T, phi::GPUContext>(
        dev_ctx, softmax_2d, {-1}, softmax_2d.dtype(), true, &sum_exp_logits);

    comm_ctx->AllReduce(&sum_exp_logits, sum_exp_logits, ncclSum, stream);

    if (label_type == phi::DataType::INT32) {
      CaculateLoss<T, int32_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_2d.data<T>(),
                                                     predicted_logits.data<T>(),
                                                     sum_exp_logits.data<T>(),
                                                     labels->data<int32_t>(),
                                                     ignore_index,
                                                     N);
    } else {
      CaculateLoss<T, int64_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_2d.data<T>(),
                                                     predicted_logits.data<T>(),
                                                     sum_exp_logits.data<T>(),
                                                     labels->data<int64_t>(),
                                                     ignore_index,
                                                     N);
    }

    auto eigen_sum_exp_logits =
        phi::funcs::EigenMatrix<T>::From(sum_exp_logits);
    eigen_softmax.device(*dev_ctx.eigen_device()) =
        (eigen_softmax *
         eigen_sum_exp_logits.inverse().broadcast(one_by_class));
#endif
  }
};

template <typename T>
struct CSoftmaxWithCrossEntropyProcessGroupFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& dev_ctx,
                  const DenseTensor& logits_in,
                  const DenseTensor& label_in,
                  int64_t ignore_index,
                  int ring_id,
                  int rank,
                  int nranks,
                  DenseTensor* softmax,
                  DenseTensor* loss) {
    const phi::DenseTensor* logits = &logits_in;
    const phi::DenseTensor* labels = &label_in;

    const int rid = ring_id;

    auto map = distributed::ProcessGroupMapFromGid::getInstance();
    distributed::ProcessGroup* pg = map->get(rid);
    distributed::AllreduceOptions opts;
    opts.reduce_op = distributed::ReduceOp::MAX;

    // allocate memory on device.
    dev_ctx.template Alloc<T>(softmax);
    dev_ctx.template Alloc<T>(loss);

    const auto& logits_dims = logits->dims();
    const auto& labels_dims = labels->dims();

    const int axis = logits_dims.size() - 1;
    const int64_t N = phi::funcs::SizeToAxis<int64_t>(axis, logits_dims);
    const int64_t D = phi::funcs::SizeFromAxis<int64_t>(axis, logits_dims);

    phi::DenseTensor logits_2d, softmax_2d, loss_2d;
    logits_2d.ShareDataWith(*logits).Resize({N, D});
    softmax_2d.ShareDataWith(*softmax).Resize({N, D});
    loss_2d.ShareDataWith(*loss).Resize({N, 1});

    auto eigen_logits = phi::funcs::EigenMatrix<T>::From(logits_2d);
    auto eigen_softmax = phi::funcs::EigenMatrix<T>::From(softmax_2d);

    // step 1, obtain logit_max
    phi::DenseTensor logits_max;
    logits_max.Resize({N, 1});
    dev_ctx.template Alloc<T>(&logits_max);

    auto eigen_logits_max = phi::funcs::EigenMatrix<T>::From(logits_max);
    Eigen::DSizes<int, 1> along_axis(1);
    eigen_logits_max.device(*dev_ctx.eigen_device()) =
        eigen_logits.maximum(along_axis);

    pg->AllReduce(&logits_max, logits_max, opts, true, true);

    // step 2, obtain logit - logit_max
    Eigen::DSizes<int, 2> batch_by_one(N, 1);
    Eigen::DSizes<int, 2> one_by_class(1, D);

    eigen_softmax.device(*dev_ctx.eigen_device()) =
        (eigen_logits -
         eigen_logits_max.reshape(batch_by_one).broadcast(one_by_class));

    // step 3, obtain predict target
    phi::DenseTensor predicted_logits;
    predicted_logits.Resize({N, 1});
    dev_ctx.template Alloc<T>(&predicted_logits);

    auto t = phi::EigenVector<T>::Flatten(predicted_logits);
    t.device(*dev_ctx.eigen_device()) = t.constant(static_cast<T>(0));

    const int64_t start_index = rank * D;
    const int64_t end_index = start_index + D;

    int64_t blocks = NumBlocks(N);
    int threads = kNumCUDAThreads;
    const auto& label_type = labels->dtype();

    if (label_type == phi::DataType::INT32) {
      MaskLabelByIndex<T, int32_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
          predicted_logits.data<T>(),
          softmax_2d.data<T>(),
          labels->data<int32_t>(),
          static_cast<int32_t>(ignore_index),
          start_index,
          end_index,
          N,
          D,
          nranks);
    } else if (label_type == phi::DataType::INT64) {
      MaskLabelByIndex<T, int64_t><<<blocks, threads, 0, dev_ctx.stream()>>>(
          predicted_logits.data<T>(),
          softmax_2d.data<T>(),
          labels->data<int64_t>(),
          static_cast<int32_t>(ignore_index),
          start_index,
          end_index,
          N,
          D,
          nranks);
    }

    opts.reduce_op = distributed::ReduceOp::SUM;
    pg->AllReduce(&predicted_logits, predicted_logits, opts, true, true);

    // step 4, obtain exp(logit)
    eigen_softmax.device(*dev_ctx.eigen_device()) = eigen_softmax.exp();

    // step 5, obtain sum_exp_logits
    phi::DenseTensor sum_exp_logits;
    sum_exp_logits.Resize({N, 1});
    void* sum_exp_logits_buff = dev_ctx.template Alloc<T>(&sum_exp_logits);

    phi::SumKernel<T, phi::GPUContext>(
        dev_ctx, softmax_2d, {-1}, softmax_2d.dtype(), true, &sum_exp_logits);

    opts.reduce_op = distributed::ReduceOp::SUM;
    pg->AllReduce(&sum_exp_logits, sum_exp_logits, opts, true, true);

    if (label_type == phi::DataType::INT32) {
      CaculateLoss<T, int32_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_2d.data<T>(),
                                                     predicted_logits.data<T>(),
                                                     sum_exp_logits.data<T>(),
                                                     labels->data<int32_t>(),
                                                     ignore_index,
                                                     N);
    } else {
      CaculateLoss<T, int64_t>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_2d.data<T>(),
                                                     predicted_logits.data<T>(),
                                                     sum_exp_logits.data<T>(),
                                                     labels->data<int64_t>(),
                                                     ignore_index,
                                                     N);
    }

    auto eigen_sum_exp_logits =
        phi::funcs::EigenMatrix<T>::From(sum_exp_logits);
    eigen_softmax.device(*dev_ctx.eigen_device()) =
        (eigen_softmax *
         eigen_sum_exp_logits.inverse().broadcast(one_by_class));
  }
};

}  // namespace phi

PD_REGISTER_KERNEL(c_softmax_with_cross_entropy,
                   GPU,
                   ALL_LAYOUT,
                   phi::CSoftmaxWithCrossEntropyKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
