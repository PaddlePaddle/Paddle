#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#ifdef PADDLE_WITH_HIP
#include <hip/hip_runtime.h>
#include <hiprand_kernel.h>
#else
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/hostdevice.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/graph_sample_neighbors_kernel.h"

namespace phi {

template <typename T>
struct DegreeFunctor {
  const T* col_ptr;
  int64_t len_col_ptr;
  HOSTDEVICE explicit inline DegreeFunctor(const T* x, int64_t len_col_ptr) {
    this->col_ptr = x;
    this->len_col_ptr = len_col_ptr;
  }
  HOSTDEVICE inline int operator()(T i) const {
    return i > len_col_ptr - 1 ? 0 : col_ptr[i + 1] - col_ptr[i];
  }
};

struct MaxFunctor {
  int cap;
  HOSTDEVICE explicit inline MaxFunctor(int cap) { this->cap = cap; }
  HOSTDEVICE inline int operator()(int x) const {
    if (x > cap) {
      return cap;
    }
    return x >= 0 ? x : 0;
  }
};

template <typename T, int CTA_SIZE, int BLOCK_CTAS, int TILE_SIZE>
__global__ void SampleKernel(const uint64_t rand_seed,
                             int k,
                             const int64_t num_nodes,
                             const int64_t len_col_ptr,
                             const T* nodes,
                             const T* row,
                             const T* col_ptr,
                             const T* eids,
                             T* output,
                             T* output_eids,
                             int* output_ptr,
                             bool return_eids) {
  assert(blockDim.x == CTA_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_nodes);
#ifdef PADDLE_WITH_HIP
  hiprandState rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x,
               threadIdx.y * CTA_SIZE + threadIdx.x,
               0,
               &rng);
#else
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x,
              threadIdx.y * CTA_SIZE + threadIdx.x,
              0,
              &rng);
#endif

  while (out_row < last_row) {
    T node = nodes[out_row];
    if (node > len_col_ptr - 1) {
      out_row += BLOCK_CTAS;
      continue;
    }
    T in_row_start = col_ptr[node];
    int deg = col_ptr[node + 1] - in_row_start;
    int out_row_start = output_ptr[out_row];

    if (deg <= k) {
      for (int idx = threadIdx.x; idx < deg; idx += CTA_SIZE) {
        output[out_row_start + idx] = row[in_row_start + idx];
        if (return_eids) {
          output_eids[out_row_start + idx] = eids[in_row_start + idx];
        }
      }
    } else {
      for (int idx = threadIdx.x; idx < k; idx += CTA_SIZE) {
        output[out_row_start + idx] = idx;
      }
#ifdef PADDLE_WITH_CUDA
      __syncthreads();
#endif

      for (int idx = k + threadIdx.x; idx < deg; idx += CTA_SIZE) {
#ifdef PADDLE_WITH_HIP
        const int num = hiprand(&rng) % (idx + 1);
#else
        const int num = hiprand(&rng) % (idx + 1);
#endif
        if (num < k) {
          atomicMax(reinterpret_cast<unsigned int*>(  // NOLINT
                        output + out_row_start + num),
                    static_cast<unsigned int>(idx));  // NOLINT
        }
      }
#ifdef PADDLE_WITH_CUDA
      __syncthreads();
#endif

      for (int idx = threadIdx.x; idx < k; idx += CTA_SIZE) {
        T perm_idx = output[out_row_start + idx] + in_row_start;
        output[out_row_start + idx] = row[perm_idx];
        if (return_eids) {
          output_eids[out_row_start + idx] = eids[perm_idx];
        }
      }
    }

    out_row += BLOCK_CTAS;
  }
}

template <typename T, typename Context>
int GetTotalSampleNum(const thrust::device_ptr<const T> input,
                      const T* col_ptr,
                      int64_t len_col_ptr,
                      thrust::device_ptr<int> output_count,
                      int sample_size,
                      int bs) {
  thrust::transform(
      input, input + bs, output_count, DegreeFunctor<T>(col_ptr, len_col_ptr));
  if (sample_size >= 0) {
    thrust::transform(
        output_count, output_count + bs, output_count, MaxFunctor(sample_size));
  }
  int total_sample_num = thrust::reduce(output_count, output_count + bs);
  return total_sample_num;
}

template <typename T, typename Context>
void SampleNeighbors(const Context& dev_ctx,
                     const T* row,
                     const T* col_ptr,
                     const T* eids,
                     const thrust::device_ptr<const T> input,
                     thrust::device_ptr<T> output,
                     thrust::device_ptr<int> output_count,
                     thrust::device_ptr<T> output_eids,
                     int sample_size,
                     int bs,
                     int total_sample_num,
                     int64_t len_col_ptr,
                     bool return_eids) {
  thrust::device_vector<int> output_ptr;
  output_ptr.resize(bs);
  thrust::exclusive_scan(
      output_count, output_count + bs, output_ptr.begin(), 0);

  constexpr int CTA_SIZE = 128;
  constexpr int BLOCK_CTAS = 128 / CTA_SIZE;
  constexpr int TILE_SIZE = BLOCK_CTAS;
  const dim3 block(CTA_SIZE, BLOCK_CTAS);
  const dim3 grid((bs + TILE_SIZE - 1) / TILE_SIZE);
  SampleKernel<T, CTA_SIZE, BLOCK_CTAS, TILE_SIZE>
      <<<grid, block, 0, dev_ctx.stream()>>>(
          0,
          sample_size,
          bs,
          len_col_ptr,
          thrust::raw_pointer_cast(input),
          row,
          col_ptr,
          eids,
          thrust::raw_pointer_cast(output),
          thrust::raw_pointer_cast(output_eids),
          thrust::raw_pointer_cast(output_ptr.data()),
          return_eids);
}

template <typename T, int CTA_SIZE, int BLOCK_CTAS, int TILE_SIZE>
__global__ void FisherYatesSampleKernel(const uint64_t rand_seed,
                                        int k,
                                        const int64_t num_rows,
                                        const int64_t len_col_ptr,
                                        const T* in_rows,
                                        T* src,
                                        const T* dst_count) {
  assert(blockDim.x == CTA_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);
#ifdef PADDLE_WITH_HIP
  hiprandState rng;
  hiprand_init(
      rand_seed * gridDim.x + blockIdx.x, threadIdx.y + threadIdx.x, 0, &rng);
#else
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(
      rand_seed * gridDim.x + blockIdx.x, threadIdx.y + threadIdx.x, 0, &rng);
#endif

  while (out_row < last_row) {
    const T row = in_rows[out_row];
    if (row > len_col_ptr - 1) {
      out_row += BLOCK_CTAS;
      continue;
    }
    const T in_row_start = dst_count[row];
    const int deg = dst_count[row + 1] - in_row_start;
    int split;
    if (k < deg) {
      if (deg < 2 * k) {
        split = k;
      } else {
        split = deg - k;
      }
      for (int idx = split + threadIdx.x; idx <= deg - 1; idx += CTA_SIZE) {
#ifdef PADDLE_WITH_HIP
        const int num = hiprand(&rng) % (idx + 1);
#else
        const int num = hiprand(&rng) % (idx + 1);
#endif
        src[in_row_start + idx] = static_cast<T>(
            atomicExch(reinterpret_cast<unsigned long long int*>(  // NOLINT
                           src + in_row_start + num),
                       static_cast<unsigned long long int>(  //  NOLINT
                           src[in_row_start + idx])));
      }
#ifdef PADDLE_WITH_CUDA
      __syncthreads();
#endif
    }
    out_row += BLOCK_CTAS;
  }
}

template <typename T, int CTA_SIZE, int BLOCK_CTAS, int TILE_SIZE>
__global__ void GatherEdge(int k,
                           int64_t num_rows,
                           const T* in_rows,
                           const T* src,
                           const T* dst_count,
                           const T* eids,
                           T* outputs,
                           T* output_eids,
                           int* output_ptr,
                           T* perm_data,
                           bool return_eids) {
  assert(blockDim.x == CTA_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  while (out_row < last_row) {
    const T row = in_rows[out_row];
    const T in_row_start = dst_count[row];
    const int deg = dst_count[row + 1] - in_row_start;
    const T out_row_start = output_ptr[out_row];

    if (deg <= k) {
      for (int idx = threadIdx.x; idx < deg; idx += CTA_SIZE) {
        outputs[out_row_start + idx] = src[in_row_start + idx];
        if (return_eids) {
          output_eids[out_row_start + idx] = eids[in_row_start + idx];
        }
      }
    } else {
      int split = k;
      int begin, end;
      if (deg < 2 * k) {
        begin = 0;
        end = k;
      } else {
        begin = deg - k;
        end = deg;
      }

      for (int idx = begin + threadIdx.x; idx < end; idx += CTA_SIZE) {
        outputs[out_row_start + idx - begin] =
            src[perm_data[in_row_start + idx]];
        if (return_eids) {
          output_eids[out_row_start + idx - begin] =
              eids[perm_data[in_row_start + idx]];
        }
      }
    }
    out_row += BLOCK_CTAS;
  }
}

template <typename T, typename Context>
void FisherYatesSampleNeighbors(const Context& dev_ctx,
                                const T* row,
                                const T* col_ptr,
                                const T* eids,
                                T* perm_data,
                                const thrust::device_ptr<const T> input,
                                thrust::device_ptr<T> output,
                                thrust::device_ptr<int> output_count,
                                thrust::device_ptr<T> output_eids,
                                int sample_size,
                                int bs,
                                int total_sample_num,
                                int64_t len_col_ptr,
                                bool return_eids) {
  thrust::device_vector<int> output_ptr;
  output_ptr.resize(bs);
  thrust::exclusive_scan(
      output_count, output_count + bs, output_ptr.begin(), 0);

  constexpr int CTA_SIZE = 128;
  constexpr int BLOCK_CTAS = 128 / CTA_SIZE;
  constexpr int TILE_SIZE = BLOCK_CTAS;
  const dim3 block(CTA_SIZE, BLOCK_CTAS);
  const dim3 grid((bs + TILE_SIZE - 1) / TILE_SIZE);

  FisherYatesSampleKernel<T, CTA_SIZE, BLOCK_CTAS, TILE_SIZE>
      <<<grid, block, 0, dev_ctx.stream()>>>(0,
                                             sample_size,
                                             bs,
                                             len_col_ptr,
                                             thrust::raw_pointer_cast(input),
                                             perm_data,
                                             col_ptr);

  GatherEdge<T, CTA_SIZE, BLOCK_CTAS, TILE_SIZE>
      <<<grid, block, 0, dev_ctx.stream()>>>(
          sample_size,
          bs,
          thrust::raw_pointer_cast(input),
          row,
          col_ptr,
          eids,
          thrust::raw_pointer_cast(output),
          thrust::raw_pointer_cast(output_eids),
          thrust::raw_pointer_cast(output_ptr.data()),
          perm_data,
          return_eids);
}

template <typename T, typename Context>
void GraphSampleNeighborsKernel(
    const Context& dev_ctx,
    const DenseTensor& row,
    const DenseTensor& col_ptr,
    const DenseTensor& x,
    const paddle::optional<DenseTensor>& eids,
    const paddle::optional<DenseTensor>& perm_buffer,
    int sample_size,
    bool return_eids,
    bool flag_perm_buffer,
    DenseTensor* out,
    DenseTensor* out_count,
    DenseTensor* out_eids) {
  auto* row_data = row.data<T>();
  auto* col_ptr_data = col_ptr.data<T>();
  auto* x_data = x.data<T>();
  int bs = x.dims()[0];
  int64_t len_col_ptr = col_ptr.dims()[0];

  const thrust::device_ptr<const T> input(x_data);

  out_count->Resize({bs});
  int* out_count_data = dev_ctx.template Alloc<int>(out_count);
  thrust::device_ptr<int> output_count(out_count_data);

  int total_sample_size = GetTotalSampleNum<T, Context>(
      input, col_ptr_data, len_col_ptr, output_count, sample_size, bs);

  out->Resize({static_cast<int>(total_sample_size)});
  T* out_data = dev_ctx.template Alloc<T>(out);
  thrust::device_ptr<T> output(out_data);

  if (return_eids) {
    auto* eids_data = eids.get_ptr()->data<T>();
    out_eids->Resize({static_cast<int>(total_sample_size)});
    T* out_eids_data = dev_ctx.template Alloc<T>(out_eids);
    thrust::device_ptr<T> output_eids(out_eids_data);
    if (!flag_perm_buffer) {
      SampleNeighbors<T, Context>(dev_ctx,
                                  row_data,
                                  col_ptr_data,
                                  eids_data,
                                  input,
                                  output,
                                  output_count,
                                  output_eids,
                                  sample_size,
                                  bs,
                                  total_sample_size,
                                  len_col_ptr,
                                  return_eids);
    } else {
      DenseTensor perm_buffer_out(perm_buffer->type());
      const auto* p_perm_buffer = perm_buffer.get_ptr();
      perm_buffer_out.ShareDataWith(*p_perm_buffer);
      T* perm_buffer_out_data = perm_buffer_out.template data<T>();
      FisherYatesSampleNeighbors<T, Context>(dev_ctx,
                                             row_data,
                                             col_ptr_data,
                                             eids_data,
                                             perm_buffer_out_data,
                                             input,
                                             output,
                                             output_count,
                                             output_eids,
                                             sample_size,
                                             bs,
                                             total_sample_size,
                                             len_col_ptr,
                                             return_eids);
    }
  } else {
    // How to set null value for output_eids(thrust::device_ptr<T>)?
    // We use `output` to fill the position of unused output_eids.
    if (!flag_perm_buffer) {
      SampleNeighbors<T, Context>(dev_ctx,
                                  row_data,
                                  col_ptr_data,
                                  nullptr,
                                  input,
                                  output,
                                  output_count,
                                  output,
                                  sample_size,
                                  bs,
                                  total_sample_size,
                                  len_col_ptr,
                                  return_eids);
    } else {
      DenseTensor perm_buffer_out(perm_buffer->type());
      const auto* p_perm_buffer = perm_buffer.get_ptr();
      perm_buffer_out.ShareDataWith(*p_perm_buffer);
      T* perm_buffer_out_data = perm_buffer_out.template data<T>();
      FisherYatesSampleNeighbors<T, Context>(dev_ctx,
                                             row_data,
                                             col_ptr_data,
                                             nullptr,
                                             perm_buffer_out_data,
                                             input,
                                             output,
                                             output_count,
                                             output,
                                             sample_size,
                                             bs,
                                             total_sample_size,
                                             len_col_ptr,
                                             return_eids);
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(graph_sample_neighbors,
                   GPU,
                   ALL_LAYOUT,
                   phi::GraphSampleNeighborsKernel,
                   int,
                   int64_t) {}
