#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifndef PADDLE_WITH_HIP  // HIP not support cusolver

#include <math.h>
#include <algorithm>
#include <complex>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/slice.h"
#include "paddle/phi/kernels/impl/lstsq_kernel_impl.h"
#include "paddle/phi/kernels/impl/qr_kernel_impl.h"
#include "paddle/phi/kernels/impl/tril_triu_kernel_impl.h"
#include "paddle/phi/kernels/lstsq_kernel.h"
#include "paddle/phi/kernels/matmul_kernel.h"
#include "paddle/phi/kernels/transpose_kernel.h"
#include "paddle/phi/kernels/triangular_solve_kernel.h"

namespace phi {

enum class LapackDriverType : int { Gels, Gelsd, Gelsy, Gelss };

template <typename T, typename Context>
void LstsqKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const DenseTensor& y,
                 const Scalar& rcond_scalar,
                 const std::string& driver_string,
                 DenseTensor* solution,
                 DenseTensor* residuals,
                 DenseTensor* rank,
                 DenseTensor* singular_values) {
  auto x_dims = x.dims();
  auto y_dims = y.dims();
  int dim_size = x_dims.size();
  int m = x_dims[dim_size - 2];
  int n = x_dims[dim_size - 1];
  int nrhs = y_dims[dim_size - 1];
  int min_mn = std::min(m, n);
  int max_mn = std::max(m, n);
  int k = min_mn;

  int x_stride = phi::GetMatrixStride(x_dims);
  int y_stride = phi::GetMatrixStride(y_dims);
  int tau_stride = min_mn;
  int batch_count = phi::GetBatchCount(x_dims);

  T rcond = rcond_scalar.to<T>();

  DenseTensor* new_x = new DenseTensor();
  new_x->Resize(phi::make_ddim({batch_count, m, n}));
  dev_ctx.template Alloc<T>(new_x);
  phi::Copy<Context>(dev_ctx, x, dev_ctx.GetPlace(), true, new_x);

  DenseTensor* new_y = new DenseTensor();
  new_y->Resize(phi::make_ddim({batch_count, m, nrhs}));
  dev_ctx.template Alloc<T>(new_y);
  phi::Copy<Context>(dev_ctx, y, dev_ctx.GetPlace(), true, new_y);

  // Prepare tau
  auto tau_dims_vec = phi::vectorize<int>(x_dims);
  tau_dims_vec.pop_back();
  tau_dims_vec[tau_dims_vec.size() - 1] = min_mn;

  DenseTensor* tau = new DenseTensor();
  tau->Resize(phi::make_ddim(tau_dims_vec));
  auto tau_data = dev_ctx.template Alloc<T>(tau);

  if (m >= n) {
    DenseTensor tmp_x = phi::TransposeLast2Dim<T>(dev_ctx, *new_x);
    DenseTensor tmp_y = phi::TransposeLast2Dim<T>(dev_ctx, *new_y);
    auto x_data = tmp_x.data<T>();
    auto y_data = tmp_y.data<T>();

    // step 1, compute QR factorization using geqrf
    BatchedGeqrf<Context, T>(
        dev_ctx, batch_count, m, n, x_data, m, tau_data, x_stride, tau_stride);

    // Step 2, Y <- Q^H Y
    BatchedOrmqr<Context, T>(dev_ctx,
                             true,
                             true,
                             batch_count,
                             m,
                             nrhs,
                             k,
                             x_data,
                             x_stride,
                             tau_data,
                             tau_stride,
                             y_data,
                             y_stride);

    DenseTensor trans_r = phi::TransposeLast2Dim<T>(dev_ctx, tmp_x);
    DenseTensor slice_r =
        phi::funcs::Slice<T>(dev_ctx, trans_r, {-2}, {0}, {min_mn});
    DenseTensor* res_r = new DenseTensor();
    res_r->Resize(phi::make_ddim({batch_count, min_mn, min_mn}));
    dev_ctx.template Alloc<T>(res_r);
    phi::TrilTriuKernel<T>(dev_ctx, slice_r, 0, false, res_r);

    DenseTensor trans_y = phi::TransposeLast2Dim<T>(dev_ctx, tmp_y);
    DenseTensor slice_y =
        phi::funcs::Slice<T>(dev_ctx, trans_y, {-2}, {0}, {min_mn});

    // Step 3, solve R X = Y
    phi::TriangularSolveKernel<T, Context>(
        dev_ctx, *res_r, slice_y, true, false, false, solution);

  } else {
    auto x_data = dev_ctx.template Alloc<T>(new_x);
    auto y_data = dev_ctx.template Alloc<T>(new_y);

    // step 1, compute QR factorization using geqrf
    BatchedGeqrf<Context, T>(
        dev_ctx, batch_count, n, m, x_data, n, tau_data, x_stride, tau_stride);

    // Step 2, solve R^H Z = Y
    DenseTensor trans_r = phi::TransposeLast2Dim<T>(dev_ctx, *new_x);
    DenseTensor slice_r =
        phi::funcs::Slice<T>(dev_ctx, trans_r, {-2}, {0}, {min_mn});
    DenseTensor* res_r = new DenseTensor();
    res_r->Resize(phi::make_ddim({batch_count, min_mn, min_mn}));
    dev_ctx.template Alloc<T>(res_r);
    phi::TrilTriuKernel<T>(dev_ctx, slice_r, 0, false, res_r);

    phi::TriangularSolveKernel<T, Context>(
        dev_ctx, *res_r, *new_y, true, true, false, solution);

    // Step 3, X <- Q Z
    BatchedOrgqr<Context, T>(dev_ctx,
                             batch_count,
                             n,
                             m,
                             min_mn,
                             x_data,
                             n,
                             tau_data,
                             x_stride,
                             tau_stride);

    DenseTensor trans_q = phi::TransposeLast2Dim<T>(dev_ctx, *new_x);
    DenseTensor slice_q =
        phi::funcs::Slice<T>(dev_ctx, trans_q, {-1}, {0}, {m});
    DenseTensor solu_tensor =
        phi::Matmul<T>(dev_ctx, slice_q, *solution, false, false);
    phi::Copy<Context>(
        dev_ctx, solu_tensor, dev_ctx.GetPlace(), true, solution);
  }

  if (batch_count == 1) solution->Resize(phi::make_ddim({n, nrhs}));
  GetResidualsTensor<Context, T>(dev_ctx, x, y, solution, residuals);
}

}  // namespace phi

PD_REGISTER_KERNEL(lstsq,  // cuda_only
                   GPU,
                   ALL_LAYOUT,
                   phi::LstsqKernel,
                   float,
                   double) {}

#endif  // not PADDLE_WITH_HIP
