#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/index_sample_grad_kernel.h"

#include <algorithm>
#include <vector>
#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

namespace {
template <typename Context>
void LimitGridDim(const Context& ctx, dim3* grid_dim) {
  auto max_grid_dim =
      reinterpret_cast<const phi::GPUContext&>(ctx).GetCUDAMaxGridDimSize();
  grid_dim->x = grid_dim->x < max_grid_dim[0] ? grid_dim->x : max_grid_dim[0];
  grid_dim->y = grid_dim->y < max_grid_dim[1] ? grid_dim->y : max_grid_dim[1];
}
#define PREDEFINED_BLOCK_SIZE_X 512
#define PREDEFINED_BLOCK_SIZE 1024
#define MIN(a, b) ((a) < (b) ? (a) : (b))
};

template <typename T, typename IndexT = int>
__global__ void IndexSampleGrad(const IndexT* index,
                                T* in_grad,
                                const T* out_grad,
                                size_t index_length,
                                size_t input_length,
                                size_t batch_size,
                                bool same_data_in_row = true) {
  unsigned int index_i = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int index_j = blockDim.y * blockIdx.y + threadIdx.y;

  for (; index_j < batch_size; index_j += blockDim.y * gridDim.y) {
    index_i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; index_i < index_length; index_i += blockDim.x * gridDim.x) {
      unsigned int index_idx = index_j * index_length + index_i;
      unsigned int in_idx = index_j * input_length + index_i;
      IndexT sample_idx = index[index_idx];
      if (same_data_in_row) {
        paddle::platform::CudaAtomicAdd(
            &(in_grad[in_idx - index_i + sample_idx]), out_grad[sample_idx]);
      } else {
        in_grad[in_idx - index_i + sample_idx] = out_grad[index_idx];
      }
    }
  }
}

template <typename T, typename Context>
void IndexSampleGradKernel(const Context& ctx,
                           const DenseTensor& out_grad,
                           const DenseTensor& x,
                           const DenseTensor& index,
                           DenseTensor* x_grad) {
  const T* output_grad_data = out_grad.data<T>();
  T* input_grad_data = ctx.template Alloc<T>(x_grad);
  auto index_type = index.dtype();
  bool index_type_match =
      index_type == DataType::INT32 || index_type == DataType::INT64;
  PADDLE_ENFORCE_EQ(
      index_type_match,
      true,
      errors::InvalidArgument(
          "Input(Index) holds the wrong type, it holds %s, but "
          "desires to be %s or %s",
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType(index_type)),
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType(DataType::INT32)),
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType((DataType::INT64)))));

  auto stream = reinterpret_cast<const phi::GPUContext&>(ctx).stream();
  auto input_num = x.numel();
  auto input_dim = x.dims();
  auto index_dim = index.dims();
  size_t batch_size = index_dim[0];
  size_t input_length = input_dim[1];
  size_t index_length = index_dim[1];
  bool same_data_in_index_row = index_length == 1 ? false : true;

  auto block_width = paddle::platform::RoundToPowerOfTwo(index_length);
  block_width = MIN(block_width, PREDEFINED_BLOCK_SIZE_X);
  auto block_height =
      paddle::platform::RoundToPowerOfTwo(index_length * batch_size) /
      block_width;
  block_height = MIN(block_height, PREDEFINED_BLOCK_SIZE / block_width);
  dim3 block_dim(block_width, block_height);
  dim3 grid_dim((index_length + block_dim.x - 1) / block_dim.x,
                (batch_size + block_dim.y - 1) / block_dim.y);
  LimitGridDim(ctx, &grid_dim);

  phi::funcs::SetConstant<Context, T> set_zero;
  set_zero(ctx, x_grad, static_cast<T>(0));

  if (index_type == DataType::INT64) {
    const int64_t* index_data = index.data<int64_t>();
    IndexSampleGrad<T, int64_t><<<grid_dim, block_dim, 0, stream>>>(
        index_data,
        input_grad_data,
        output_grad_data,
        index_length,
        input_length,
        batch_size,
        same_data_in_index_row);
  } else if (index_type == DataType::INT32) {
    const int* index_data = index.data<int>();
    IndexSampleGrad<T, int><<<grid_dim, block_dim, 0, stream>>>(
        index_data,
        input_grad_data,
        output_grad_data,
        index_length,
        input_length,
        batch_size,
        same_data_in_index_row);
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(index_sample_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::IndexSampleGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
