#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"

#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/memory_utils.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/partial_concat_funcs.h"
#include "paddle/phi/kernels/funcs/strided_memcpy.h"

namespace phi {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

template <class T>
__global__ void ConcatPartialCUDAKernel(T **in,
                                        T *out,
                                        int64_t all_length,
                                        int64_t in_batch_len,
                                        int64_t start_index,
                                        int64_t out_batch_len,
                                        int64_t part_length) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < all_length) {
    int64_t bs_id = id / out_batch_len;
    int64_t bs_index = id % out_batch_len;
    int64_t var_id = bs_index / part_length;
    int64_t part_index = bs_index % part_length;
    int64_t in_id = start_index + part_index;
    const T *tmp = in[var_id];
    out[id] = tmp[bs_id * in_batch_len + in_id];
    id += blockDim.x * gridDim.x;
  }
}

template <typename T, typename Context>
void PartialConcatOpCUDAKernel(const Context &dev_ctx,
                               const std::vector<const DenseTensor *> &x,
                               int start_index,
                               int length,
                               DenseTensor *out) {
  auto in_vars = x;
  PADDLE_ENFORCE_EQ(in_vars[0] != nullptr,
                    true,
                    phi::errors::InvalidArgument(
                        "The input of partial concat should not be null."));

  auto input_dim = in_vars[0]->dims();
  PADDLE_ENFORCE_EQ(input_dim.size(),
                    2,
                    phi::errors::InvalidArgument(
                        "Only supports 2-D array with batch size in the 1st "
                        "dimension and data in the 2nd."));
  auto in_size = input_dim[1];
  // may be negative
  start_index = ComputeStartIndex(start_index, in_size);

  auto partial_len = length;
  if (partial_len < 0) {
    partial_len = in_size - start_index;
  }

  int in_num = in_vars.size();
  int batch_size = input_dim[0];
  int out_batch_len = partial_len * in_num;
  int all_length = batch_size * out_batch_len;

  constexpr size_t theory_sm_threads = 1024;
  auto stream = dev_ctx.stream();
  auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  auto sm_count = max_threads / theory_sm_threads;
  size_t tile_size = 0;
  int grids;
  int blocks;
  auto ComputeKernelParameter = [&](size_t length) {
    if (length >= max_threads)
      tile_size = 1024;
    else if (length < max_threads && length > sm_count * 128)
      tile_size = 512;
    else if (length <= sm_count * 128)
      tile_size = 256;
    grids = CEIL_DIV(length, tile_size);
    blocks = tile_size;
  };

  T *out_data = dev_ctx.template Alloc<T>(out);

  std::vector<const T *> in_data;
  for (int i = 0; i < in_num; ++i) in_data.emplace_back(in_vars[i]->data<T>());

  auto tmp_in_array = phi::memory_utils::Alloc(
      dev_ctx.GetPlace(),
      in_data.size() * sizeof(T *),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  phi::memory_utils::Copy(dev_ctx.GetPlace(),
                          tmp_in_array->ptr(),
                          phi::CPUPlace(),
                          reinterpret_cast<void *>(in_data.data()),
                          in_data.size() * sizeof(T *),
                          dev_ctx.stream());

  T **in_array_data = reinterpret_cast<T **>(tmp_in_array->ptr());
  ComputeKernelParameter(all_length);
  ConcatPartialCUDAKernel<T><<<grids, blocks, 0, stream>>>(in_array_data,
                                                           out->data<T>(),
                                                           all_length,
                                                           in_size,
                                                           start_index,
                                                           out_batch_len,
                                                           partial_len);
}
}  // namespace phi

PD_REGISTER_KERNEL(partial_concat,
                   GPU,
                   ALL_LAYOUT,
                   phi::PartialConcatOpCUDAKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
