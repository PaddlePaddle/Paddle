#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#ifndef PADDLE_WITH_HIP
// To-do(qili93): fix this after issue resolved
// https://github.com/ROCmSoftwarePlatform/rocPRIM/issues/202

#include "paddle/phi/kernels/multinomial_kernel.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/scalar.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/arg_min_max_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/distribution_helper.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/funcs/inclusive_scan.h"
#include "paddle/phi/kernels/funcs/multinomial_functor.h"
#include "paddle/phi/kernels/top_k_kernel.h"

namespace phi {

template <typename T>
__global__ void NormalizeProbability(T* norm_probs,
                                     const T* in_data,
                                     T* sum_rows,
                                     int64_t num_distributions,
                                     int64_t num_categories) {
  int id = threadIdx.x + blockIdx.x * blockDim.x +
           blockIdx.y * gridDim.x * blockDim.x;
  if (id < num_distributions * num_categories) {
    PADDLE_ENFORCE(
        in_data[id] >= 0.0,
        "The input of multinomial distribution should be >= 0, but got %f.",
        in_data[id]);
    int64_t row_id = id / num_categories;
    PADDLE_ENFORCE(sum_rows[row_id] > 0.0,
                   "The sum of one multinomial distribution probability should "
                   "be > 0, but got %f.",
                   sum_rows[row_id]);
    norm_probs[id] = in_data[id] / sum_rows[row_id];
  }
}

template <typename T>
__device__ int binarySearchFunctor(T* cumulative_probs_data,
                                   T* norm_probs_data,
                                   int num_categories,
                                   T rng_number) {
  int left = 0;
  int right = num_categories;

  while (right - left > 0) {
    int mid = left + (right - left) / 2;

    T temp_prob = cumulative_probs_data[mid];
    if (temp_prob < rng_number) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }

  if (left == num_categories) {
    left = num_categories - 1;
  }

  while (left >= 1 && norm_probs_data[left] == 0) left--;

  return left;
}

template <typename T>
__global__ void sampleMultinomialWithReplacement(
    const int64_t num_samples,
    int64_t* out_data,
    const int64_t num_distributions,
    const int64_t num_categories,
    T* cumulative_probs_data,
    T* norm_probs_data,
    uint64_t seed,
    uint64_t offset) {
  // use binary search to get the selected category sample id.
  // let cumulative_probs_data[id-1] < rng_number < cumulative_probs_data[id].
  size_t idx = gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x +
               threadIdx.x;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);

  int sample = blockIdx.x * blockDim.x + threadIdx.x;
  for (int dist = blockIdx.y; dist < num_distributions; dist += gridDim.y) {
    if (sample < num_samples) {
      T rng_number = static_cast<T>(hiprand_uniform4(&state).x);
      // Find the bucket that a uniform random number lies in
      int selected_category =
          binarySearchFunctor<T>(cumulative_probs_data + dist * num_categories,
                                 norm_probs_data + dist * num_categories,
                                 num_categories,
                                 rng_number);

      out_data[sample + dist * num_samples] = selected_category;
    }
  }
}

template <typename T, typename Context>
void MultinomialKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const Scalar& num_samples,
                       bool replacement,
                       DenseTensor* out) {
  auto int_num_samples = num_samples.to<int>();
  auto* in_data = x.data<T>();
  int64_t* out_data = dev_ctx.template Alloc<int64_t>(out);
  auto in_dims = x.dims();
  int64_t dim_size = in_dims.size();
  const int64_t num_categories = in_dims[dim_size - 1];
  const int64_t num_distributions = dim_size > 1 ? in_dims[dim_size - 2] : 1;

  // If replacement is False, it's not a replaceable sample. Every category
  // can be used only once.
  if (!replacement) {
    int64_t in_data_numel = x.numel();
    int64_t out_data_numel = out->numel();

    // Just use to PADDLE_ENFORCE error message
    T* cpu_in_data = new T[in_data_numel];

#ifdef PADDLE_WITH_HIP
    hipMemcpy(
        cpu_in_data, in_data, in_data_numel * sizeof(T), hipMemcpyDeviceToHost);
#else
    hipMemcpy(cpu_in_data,
               in_data,
               in_data_numel * sizeof(T),
               hipMemcpyDeviceToHost);
#endif
    for (size_t i = 0; i < num_distributions; ++i) {
      int zero_num = 0;
      for (size_t j = 0; j < num_categories; ++j) {
        T weight = cpu_in_data[i * num_categories + j];
        PADDLE_ENFORCE_GE(
            weight,
            0,
            errors::InvalidArgument(
                "Each element of multinomial'input must >= 0, but got %f.",
                weight));
        if (weight == static_cast<T>(0)) {
          zero_num++;
        }
      }
      int valid_samples = num_categories - zero_num;
      PADDLE_ENFORCE_LE(
          int_num_samples,
          valid_samples,
          errors::InvalidArgument("When replacement=False, 'num_samples' "
                                  "must less than or eaqual to the number of "
                                  "positive item of input"));
    }

    // Refer to [gumbel softmax algorithm]
    DenseTensor rand = EmptyLike<T, Context>(dev_ctx, x);
    T* rand_data = rand.data<T>();
    funcs::uniform_distribution<T> dist;
    funcs::exponential_transform<T> trans(1.0);
    funcs::distribution_and_transform<T>(dev_ctx, &rand, dist, trans);

    funcs::ForRange<Context> for_range(dev_ctx, x.numel());
    for_range([rand_data, in_data] __device__(size_t idx) {
      rand_data[idx] = in_data[idx] / rand_data[idx];
    });

    if (int_num_samples == 1) {
      ArgMaxKernel<T, Context>(
          dev_ctx, rand, -1, true, false, 3 /*proto::VarType::INT64*/, out);
    } else {
      std::vector<int64_t> out_dim_vec = vectorize<int64_t>(out->dims());
      DenseTensor value = Empty<T, Context>(dev_ctx, IntArray(out_dim_vec));
      TopkKernel<T, Context>(
          dev_ctx, rand, num_samples, -1, true, true, &value, out);
    }
    return;
  }

  // Sum of input may not be 1. To get probability in range [0, 1], calculate
  // sum of each row of input, and then use the sum to normalize the input.
  // sum_row_data: sum of each row
  DenseTensor sum_rows_tensor;
  sum_rows_tensor.Resize({num_distributions});
  auto* sum_rows_data = dev_ctx.template Alloc<T>(&sum_rows_tensor);

  auto& place = *dev_ctx.eigen_device();

  if (num_distributions == 1) {
    auto eigen_input = EigenVector<T>::Flatten(x);
    auto eigen_sum_rows = EigenVector<T>::Flatten(sum_rows_tensor);
    eigen_sum_rows.device(place) =
        eigen_input.sum(Eigen::DSizes<int, 1>(1))
            .eval()
            .reshape(Eigen::DSizes<int, 1>(sum_rows_tensor.dims()[0]));
  } else {
    auto eigen_input = EigenMatrix<T>::From(x);
    auto eigen_sum_rows = EigenVector<T>::Flatten(sum_rows_tensor);
    eigen_sum_rows.device(place) = eigen_input.sum(Eigen::DSizes<int, 1>(1));
  }

  // Normalize row of each distribution to get the probability in range [0,
  // 1].
  // norm_probs_data: probability of the distribution
  DenseTensor norm_probs_tensor;
  norm_probs_tensor.Resize({num_distributions, num_categories});
  auto* norm_probs_data = dev_ctx.template Alloc<T>(&norm_probs_tensor);

  // number of threads in a block is min(num_categories, 512)
  int block_size = num_categories < 512 ? num_categories : 512;
  dim3 block_norm(block_size);
  dim3 grid_norm((num_distributions * num_categories - 1) / block_norm.x + 1);
  NormalizeProbability<T>
      <<<grid_norm, block_norm, 0, dev_ctx.stream()>>>(norm_probs_data,
                                                       in_data,
                                                       sum_rows_data,
                                                       num_distributions,
                                                       num_categories);

  // Get cumulative probability of each distribution. It's the same function
  // of ``cumsum`` op.
  DenseTensor cumulative_probs_tensor;
  cumulative_probs_tensor.Resize({num_distributions, num_categories});
  auto* cumulative_probs_data =
      dev_ctx.template Alloc<T>(&cumulative_probs_tensor);

  // 'phi::funcs::InclusiveScan' has higher accuracy than
  // 'thrust::inclusive_scan'
  funcs::InclusiveScan<T, std::plus<T>>(
      /*in*/ norm_probs_data,
      /*out*/ cumulative_probs_data,
      /*outer_dim*/ static_cast<size_t>(num_distributions),
      /*mid_dim*/ static_cast<size_t>(num_categories),
      /*inner_dim*/ static_cast<size_t>(1),
      /*init*/ static_cast<T>(0),
      std::plus<T>(),
      /*reverse=*/false,
      dev_ctx);

  // Sample the multinomial distributions.
  dim3 block(128);
  int64_t device_id = dev_ctx.GetPlace().GetDeviceId();
  const auto& prop = phi::backends::gpu::GetDeviceProperties(device_id);
  int grid_y = std::min<int64_t>(num_distributions, prop.maxGridSize[1]);
  dim3 grid((int_num_samples - 1) / block.x + 1, grid_y);

  auto gen_cuda = dev_ctx.GetGenerator();
  size_t curand4_loop_times =
      (num_distributions + 4 * grid_y - 1) / (4 * grid_y);
  // 'increment' shoulde be multiple of 4
  uint64_t increment = curand4_loop_times * 4;
  auto seed_offset = gen_cuda->IncrementOffset(increment);

  sampleMultinomialWithReplacement<T>
      <<<grid, block, 0, dev_ctx.stream()>>>(int_num_samples,
                                             out_data,
                                             num_distributions,
                                             num_categories,
                                             cumulative_probs_data,
                                             norm_probs_data,
                                             seed_offset.first,
                                             seed_offset.second);
}

}  // namespace phi

PD_REGISTER_KERNEL(multinomial,  // cuda_only
                   GPU,
                   ALL_LAYOUT,
                   phi::MultinomialKernel,
                   float,
                   double) {}

#endif
