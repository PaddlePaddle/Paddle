// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/llm_int8_mat_mul_kernel.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#if !defined(PADDLE_WITH_HIP) && !defined(PADDLE_WITH_MUSA)
#include "paddle/phi/kernels/impl/llm_int8_mat_mul_kernel_impl.h"
#endif

namespace phi {

template <typename T, typename Context>
void llm_int8_compute(const Context& dev_ctx,
                      const DenseTensor& x,
                      const DenseTensor& weight,
                      const DenseTensor& weight_scale,
                      const float threshold,
                      DenseTensor* out) {
#if defined(PADDLE_WITH_HIP)
  LOG(ERROR) << "Please compile with cublaslt, ROCM platform isn't support it";
#else
  DenseTensor cublaslt_workspace;
  cublaslt_workspace.Resize({{3000000}});
  dev_ctx.template Alloc<int8_t>(&cublaslt_workspace);
  const auto x_dims = x.dims();
  const auto w_dims = weight.dims();
  int k = w_dims[1];
  int n = w_dims[0];
  int m = x.numel() / k;
  // mk * transpose(nk) = mn
  llm_int8::LLMGemm<T>(dev_ctx,
                       &weight,
                       &x,
                       &weight_scale,
                       threshold,
                       out,
                       &cublaslt_workspace,
                       "llm_int8_mat_mul",
                       m,
                       k,
                       n);
#endif
}

template <typename T, typename Context>
void LLMInt8MatMulKernel(const Context& dev_ctx,
                         const DenseTensor& x,
                         const DenseTensor& weight,
                         const DenseTensor& weight_scale,
                         const float threshold,
                         DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  llm_int8_compute<T, Context>(
      dev_ctx, x, weight, weight_scale, threshold, out);
}
}  // namespace phi

PD_REGISTER_KERNEL(llm_int8_mat_mul,
                   GPU,
                   ALL_LAYOUT,
                   phi::LLMInt8MatMulKernel,
                   phi::dtype::float16) {}
