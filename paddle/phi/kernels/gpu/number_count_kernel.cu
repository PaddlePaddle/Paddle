#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/number_count_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {
#define CEIL(_x_, _y_) (((_x_)-1) / (_y_) + 1)
#define PERTHREAD_EXPERTS 256
#define WARP_SIZE 32

const int CUDA_NUM_THREADS = 512;
static inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename T>
__global__ void initialize_zero_kernel(T* data, const int length) {
  CUDA_KERNEL_LOOP(idx, length) { data[idx] = static_cast<T>(0); }
}

template <typename T>
__global__ void NumberCount(const T* numbers,
                            T* number_count,
                            int64_t batch_size,
                            int upper_range) {
  int res_tmp[PERTHREAD_EXPERTS] = {0};
  int expert_min = blockIdx.x * PERTHREAD_EXPERTS;
  int expert_max = expert_min + PERTHREAD_EXPERTS;
  if (expert_max > upper_range) {
    expert_max = upper_range;
  }
  for (int i = threadIdx.x; i < batch_size; i += blockDim.x) {
    T idx = numbers[i];
    if (idx == -1) {
      continue;
    }
    if (idx < expert_min || idx >= expert_max) {
      continue;
    }
    res_tmp[idx - expert_min] += 1;
  }
  for (int i = expert_min; i < expert_max; ++i) {
    int x = res_tmp[i - expert_min];
#pragma unroll
    for (int j = 1; j < WARP_SIZE; j <<= 1) {
#ifdef __HIPCC__
      x = x + __shfl_down(x, j);
#else
      x = x + __shfl_down_sync(-1u, x, j);
#endif
    }
    if (threadIdx.x % WARP_SIZE == 0) {
      phi::CudaAtomicAdd(number_count + i, x);
    }
  }
}

template <typename T, typename Context>
void NumberCountKernel(const Context& ctx,
                       const DenseTensor& numbers,
                       int upper_range,
                       DenseTensor* out) {
  int64_t batch_size = numbers.numel();

  DDim out_dims = common::make_ddim({upper_range});
  out->Resize(out_dims);
  auto out_data = ctx.template Alloc<T>(out);
  const T* gate_data = numbers.data<T>();

  initialize_zero_kernel<T>
      <<<GET_BLOCKS(upper_range), CUDA_NUM_THREADS, 0, ctx.stream()>>>(
          out_data, upper_range);

  NumberCount<T>
      <<<CEIL(upper_range, PERTHREAD_EXPERTS), 256, 0, ctx.stream()>>>(
          gate_data, out_data, batch_size, upper_range);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    number_count, GPU, ALL_LAYOUT, phi::NumberCountKernel, int64_t) {}
