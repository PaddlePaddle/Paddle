// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/cumprod_grad_kernel.h"

#include <thrust/transform.h>

#include "paddle/fluid/operators/math/inclusive_scan.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"
#include "paddle/phi/kernels/funcs/cumprod.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/funcs/for_range.h"
// NOTE(@xiongkun): use of IsComplex<>
#include "paddle/fluid/framework/data_type.h"

namespace phi {

template <typename T>
struct CumprodGradFunctorExceptFirstZero {
  HOSTDEVICE CumprodGradFunctorExceptFirstZero(
      const T *x,
      const T *y,
      const T *dy_mul_y_reversed_cumsum,
      const uint8_t *zero_mask,
      size_t mid_dim,
      size_t inner_dim,
      T *dx,
      int64_t *first_zero_idx,
      T *x_filled_one)
      : x_(x),
        y_(y),
        dy_mul_y_reversed_cumsum_(dy_mul_y_reversed_cumsum),
        zero_mask_(zero_mask),
        mid_dim_(mid_dim),
        inner_dim_(inner_dim),
        dx_(dx),
        first_zero_idx_(first_zero_idx),
        x_filled_one_(x_filled_one) {}

  HOSTDEVICE void operator()(size_t idx) const {
    auto inner_idx = idx % inner_dim_;
    auto outer_idx = idx / (mid_dim_ * inner_dim_);
    auto mid_idx = (idx - inner_idx) / inner_dim_ % mid_dim_;
    auto mask = zero_mask_[idx];
    bool should_fill_one = true;

    if (mask == 0) {
      dx_[idx] = dy_mul_y_reversed_cumsum_[idx] / x_[idx];
      if (mid_idx == mid_dim_ - 1) {
        // record first zero position as -1, i.e., no zero
        first_zero_idx_[outer_idx * inner_dim_ + inner_idx] = -1;
      }
    } else if (mid_idx > 0) {                  // mask > 0
      if (zero_mask_[idx - inner_dim_] > 0) {  // not first zero
        dx_[idx] = 0;
        should_fill_one = false;
      } else {
        // idx is the first zero position, it should be recorded
        dx_[idx] = y_[idx - inner_dim_];
        first_zero_idx_[outer_idx * inner_dim_ + inner_idx] = mid_idx;
      }
    } else {  // the first zero position is index 0
      dx_[idx] = 1;
      first_zero_idx_[outer_idx * inner_dim_ + inner_idx] = 0;
    }

    x_filled_one_[idx] = should_fill_one ? 1 : x_[idx];
  }

 private:
  const T *x_;
  const T *y_;
  const T *dy_mul_y_reversed_cumsum_;
  const uint8_t *zero_mask_;
  size_t mid_dim_;
  size_t inner_dim_;
  T *dx_;
  int64_t *first_zero_idx_;
  T *x_filled_one_;
};

template <typename T>
struct FillFirstZeroPositionGradFunctor {
  HOSTDEVICE FillFirstZeroPositionGradFunctor(const int64_t *first_zero_idx,
                                              const T *grad_value,
                                              size_t mid_dim,
                                              size_t inner_dim,
                                              T *dx)
      : first_zero_idx_(first_zero_idx),
        grad_value_(grad_value),
        mid_dim_(mid_dim),
        inner_dim_(inner_dim),
        dx_(dx) {}

  HOSTDEVICE void operator()(size_t idx) const {
    auto outer_idx = idx / inner_dim_;
    auto inner_idx = idx % inner_dim_;
    auto mid_idx = first_zero_idx_[idx];
    if (mid_idx >= 0) {
      auto full_idx =
          outer_idx * mid_dim_ * inner_dim_ + mid_idx * inner_dim_ + inner_idx;
      dx_[full_idx] *= grad_value_[full_idx];
    }
  }

 private:
  const int64_t *first_zero_idx_;
  const T *grad_value_;
  size_t mid_dim_;
  size_t inner_dim_;
  T *dx_;
};

template <typename T, typename Context>
void CumprodGradKernel(const Context &dev_ctx,
                       const DenseTensor &x,
                       const DenseTensor &out,
                       const DenseTensor &dout,
                       int dim,
                       DenseTensor *dx) {
  const auto *y = &out;
  const auto *dy = &dout;

  size_t outer_dim, mid_dim, inner_dim;
  GetCumprodDimInfo(x.dims(), dim, &outer_dim, &mid_dim, &inner_dim);
  if (outer_dim == 0 || mid_dim == 0 || inner_dim == 0) return;

  size_t numel = outer_dim * mid_dim * inner_dim;

  const auto *x_data = x.data<T>();
  const auto *y_data = y->data<T>();
  const auto *dy_data = dy->data<T>();

  auto place = dev_ctx.GetPlace();
  auto *dx_data = dev_ctx.template Alloc<T>(dx);

  // deal with complex
  const T *x_data_deal;
  const T *y_data_deal;
  Allocator::AllocationPtr x_conj;
  Allocator::AllocationPtr y_conj;
  if (paddle::framework::IsComplex<T>::value) {
    x_conj = const_cast<Allocator &>(dev_ctx.GetAllocator())
                 .Allocate(numel * sizeof(T));
    auto *x_data_conj = reinterpret_cast<T *>(x_conj->ptr());
    y_conj = const_cast<Allocator &>(dev_ctx.GetAllocator())
                 .Allocate(numel * sizeof(T));
    auto *y_data_conj = reinterpret_cast<T *>(y_conj->ptr());

    phi::funcs::ForRange<Context> for_range_x(dev_ctx, numel);
    phi::funcs::ConjFunctor<T> functor_x(x_data, numel, x_data_conj);
    for_range_x(functor_x);

    phi::funcs::ForRange<Context> for_range_y(dev_ctx, numel);
    phi::funcs::ConjFunctor<T> functor_y(y_data, numel, y_data_conj);
    for_range_y(functor_y);
    x_data_deal = x_data_conj;
    y_data_deal = y_data_conj;
  } else {
    x_data_deal = x_data;
    y_data_deal = y_data;
  }

// Step 1: find cummax-ed zero mask of x
#ifdef PADDLE_WITH_CUDA
  const auto &exec_policy = thrust::cuda::par.on(dev_ctx.stream());
#else
  const auto &exec_policy = thrust::hip::par.on(dev_ctx.stream());
#endif
  auto zero_mask_without_cummax =
      const_cast<Allocator &>(dev_ctx.GetAllocator())
          .Allocate(numel * sizeof(uint8_t));
  auto *zero_mask_without_cummax_data =
      reinterpret_cast<uint8_t *>(zero_mask_without_cummax->ptr());
  thrust::transform(exec_policy,
                    thrust::device_pointer_cast(x_data_deal),
                    thrust::device_pointer_cast(x_data_deal) + numel,
                    thrust::device_pointer_cast(zero_mask_without_cummax_data),
                    funcs::IsZeroFunctor<T>());

  auto zero_mask = const_cast<Allocator &>(dev_ctx.GetAllocator())
                       .Allocate(numel * sizeof(uint8_t));
  auto *zero_mask_data = reinterpret_cast<uint8_t *>(zero_mask->ptr());
  paddle::operators::math::InclusiveScan<uint8_t, hipcub::Max>(
      zero_mask_without_cummax_data,
      zero_mask_data,
      outer_dim,
      mid_dim,
      inner_dim,
      static_cast<uint8_t>(0),
      hipcub::Max(),
      /*reverse=*/false,
      dev_ctx);
  zero_mask_without_cummax = nullptr;

  // Step 2: calculate reversed cumsum(dy * y)
  auto dy_mul_y = const_cast<Allocator &>(dev_ctx.GetAllocator())
                      .Allocate(numel * sizeof(T));
  auto *dy_mul_y_data = reinterpret_cast<T *>(dy_mul_y->ptr());
  thrust::transform(exec_policy,
                    thrust::device_pointer_cast(dy_data),
                    thrust::device_pointer_cast(dy_data) + numel,
                    thrust::device_pointer_cast(y_data_deal),
                    thrust::device_pointer_cast(dy_mul_y_data),
                    funcs::MultiplyFunctor<T>());

  auto dy_mul_y_reversed_cumsum =
      const_cast<Allocator &>(dev_ctx.GetAllocator())
          .Allocate(numel * sizeof(T));
  auto *dy_mul_y_reversed_cumsum_data =
      reinterpret_cast<T *>(dy_mul_y_reversed_cumsum->ptr());
  paddle::operators::math::InclusiveScan<T, hipcub::Sum>(
      dy_mul_y_data,
      dy_mul_y_reversed_cumsum_data,
      outer_dim,
      mid_dim,
      inner_dim,
      static_cast<T>(0),
      hipcub::Sum(),
      /*reverse=*/true,
      dev_ctx);

  // Step 3: calculate the gradient value except the first zero position.
  // The gradient value of the first zero position is filled with out[idx-1],
  // while the gradient value of the other positions are calculated out
  // completely. This functor also:
  //  (1) find the first zero index, i.e., first_zero_idx_data.
  //  (2) fill x_filled_one, which satifies
  //      x_filled_one[i] = x[i], i > pos
  //      x_filled_one[i] = 1, i <= pos
  auto first_zero_idx = const_cast<Allocator &>(dev_ctx.GetAllocator())
                            .Allocate(numel * sizeof(int64_t));
  auto *first_zero_idx_data =
      reinterpret_cast<int64_t *>(first_zero_idx->ptr());
  auto *x_filled_one_data = dy_mul_y_data;  // reuse former allocated memory
  phi::funcs::ForRange<Context> for_range(dev_ctx, numel);
  CumprodGradFunctorExceptFirstZero<T> functor_except_first_zero(
      x_data_deal,
      y_data_deal,
      dy_mul_y_reversed_cumsum_data,
      zero_mask_data,
      mid_dim,
      inner_dim,
      dx_data,
      first_zero_idx_data,
      x_filled_one_data);
  for_range(functor_except_first_zero);

  // Step 4: calculate cumprod of x_filled_one
  auto *x_filled_one_cumprod_data =
      dy_mul_y_reversed_cumsum_data;  // reuse former allocated memory
  paddle::operators::math::InclusiveScan<T, funcs::MultiplyFunctor<T>>(
      x_filled_one_data,
      x_filled_one_cumprod_data,
      outer_dim,
      mid_dim,
      inner_dim,
      static_cast<T>(1),
      funcs::MultiplyFunctor<T>(),
      /*reverse=*/false,
      dev_ctx);

  // Step 5: calculate reversed cumsum(dy * x_filled_one_cumprod)
  auto *dy_mul_x_filled_one_cumprod =
      dy_mul_y_data;  // reuse former allocated memory
  thrust::transform(exec_policy,
                    thrust::device_pointer_cast(dy_data),
                    thrust::device_pointer_cast(dy_data) + numel,
                    thrust::device_pointer_cast(x_filled_one_cumprod_data),
                    thrust::device_pointer_cast(dy_mul_x_filled_one_cumprod),
                    funcs::MultiplyFunctor<T>());
  auto *dy_mul_x_filled_one_cumprod_reversed_cumsum =
      dy_mul_y_reversed_cumsum_data;  // reuse former allocated memory
  paddle::operators::math::InclusiveScan<T, hipcub::Sum>(
      dy_mul_x_filled_one_cumprod,
      dy_mul_x_filled_one_cumprod_reversed_cumsum,
      outer_dim,
      mid_dim,
      inner_dim,
      static_cast<T>(0),
      hipcub::Sum(),
      /*reverse=*/true,
      dev_ctx);

  // Step 6: fill zero pos gradient value
  phi::funcs::ForRange<Context> for_range_fill_zero_pos_grad(
      dev_ctx, outer_dim * inner_dim);
  FillFirstZeroPositionGradFunctor<T> fill_first_zero_pos_grad_functor(
      first_zero_idx_data,
      dy_mul_x_filled_one_cumprod_reversed_cumsum,
      mid_dim,
      inner_dim,
      dx_data);
  for_range_fill_zero_pos_grad(fill_first_zero_pos_grad_functor);
}

}  // namespace phi

PD_REGISTER_KERNEL(cumprod_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CumprodGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
