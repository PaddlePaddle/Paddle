#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/auc_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/cpu/cpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

__global__ void ClearObsoleteDataKernel(int64_t *pos,
                                        int64_t *neg,
                                        const int bucket_length,
                                        const int slide_steps) {
  int cur_step_index =
      static_cast<int>(pos[(slide_steps + 1) * bucket_length]) % slide_steps;
  int cur_step_begin = cur_step_index * bucket_length;
  int sum_step_begin = slide_steps * bucket_length;
  CUDA_KERNEL_LOOP(i, bucket_length) {
    pos[sum_step_begin + i] -= pos[cur_step_begin + i];
    neg[sum_step_begin + i] -= neg[cur_step_begin + i];
    pos[cur_step_begin + i] = neg[cur_step_begin + i] = 0;
  }
}

__global__ void UpdateSumDataKernel(int64_t *pos,
                                    int64_t *neg,
                                    const int bucket_length,
                                    const int slide_steps) {
  int cur_step_index =
      static_cast<int>(pos[(slide_steps + 1) * bucket_length]) % slide_steps;
  int cur_step_begin = cur_step_index * bucket_length;
  int sum_step_begin = slide_steps * bucket_length;
  CUDA_KERNEL_LOOP(i, bucket_length) {
    pos[sum_step_begin + i] += pos[cur_step_begin + i];
    neg[sum_step_begin + i] += neg[cur_step_begin + i];
  }
}

template <typename T>
__global__ void AddDataKernel(const int64_t *label_data,
                              const T *pred_data,
                              const int inference_width,
                              const int num_thresholds,
                              int64_t *pos,
                              int64_t *neg,
                              const int numel,
                              const int slide_steps) {
  int cur_step_begin = 0;
  if (slide_steps > 0) {
    int cur_step_index =
        static_cast<int>(pos[(slide_steps + 1) * (1 + num_thresholds)]) %
        slide_steps;
    cur_step_begin = cur_step_index * (1 + num_thresholds);
  }
  CUDA_KERNEL_LOOP(i, numel) {
    auto predict_data = pred_data[i * inference_width + (inference_width - 1)];
    PADDLE_ENFORCE(predict_data <= 1, "The predict data must less or equal 1.");
    PADDLE_ENFORCE(predict_data >= 0,
                   "The predict data must gather or equal 0.");
    uint32_t binIdx = static_cast<uint32_t>(predict_data * num_thresholds);
    if (label_data[i]) {
      paddle::platform::CudaAtomicAdd(pos + cur_step_begin + binIdx, 1);
    } else {
      paddle::platform::CudaAtomicAdd(neg + cur_step_begin + binIdx, 1);
    }
  }
}

__global__ void CalcAucKernel(int64_t *stat_pos,
                              int64_t *stat_neg,
                              int num_thresholds,
                              double *auc,
                              bool need_add_batch_num) {
  *auc = 0.0f;
  double totPos = 0.0;
  double totNeg = 0.0;
  double totPosPrev = 0.0;
  double totNegPrev = 0.0;

  int idx = num_thresholds;

  while (idx >= 0) {
    totPosPrev = totPos;
    totNegPrev = totNeg;
    totPos += stat_pos[idx];
    totNeg += stat_neg[idx];
    *auc += (totNeg - totNegPrev) * (totPos + totPosPrev) / 2.0;
    --idx;
  }

  if (totPos > 0.0 && totNeg > 0.0) {
    *auc = *auc / totPos / totNeg;
  }
  if (need_add_batch_num) {
    stat_pos[num_thresholds + 1] += 1;
    stat_neg[num_thresholds + 1] += 1;
  }
}

inline static double trapezoidArea(double X1, double X2, double Y1, double Y2) {
  return (X1 > X2 ? (X1 - X2) : (X2 - X1)) * (Y1 + Y2) / 2.0;
}

template <typename T, typename Context>
void statAuc(const Context &dev_ctx,
             const DenseTensor &label,
             const DenseTensor &predict,
             const int num_thresholds,
             const int slide_steps,
             int64_t *origin_stat_pos,
             int64_t *origin_stat_neg,
             const bool is_fake_data) {
  size_t batch_size = predict.dims()[0];
  size_t inference_width = predict.dims()[1];
  const T *inference_data = predict.data<T>();
  const auto *label_data = label.data<int64_t>();
  const int bucket_length = num_thresholds + 1;

  if (slide_steps == 0) {
    AddDataKernel<<<(batch_size + PADDLE_CUDA_NUM_THREADS - 1) /
                        PADDLE_CUDA_NUM_THREADS,
                    PADDLE_CUDA_NUM_THREADS,
                    0,
                    dev_ctx.stream()>>>(label_data,
                                        inference_data,
                                        inference_width,
                                        num_thresholds,
                                        origin_stat_pos,
                                        origin_stat_neg,
                                        batch_size,
                                        slide_steps);
    return;
  }
  // the last number of origin_stat_pos store the index should be used in
  // current step
  int cur_step_index =
      static_cast<int>(origin_stat_pos[(slide_steps + 1) * bucket_length]) %
      slide_steps;
  int cur_step_begin = cur_step_index * bucket_length;
  int sum_step_begin = slide_steps * bucket_length;

  ClearObsoleteDataKernel<<<(bucket_length + PADDLE_CUDA_NUM_THREADS - 1) /
                                PADDLE_CUDA_NUM_THREADS,
                            PADDLE_CUDA_NUM_THREADS,
                            0,
                            dev_ctx.stream()>>>(
      origin_stat_pos, origin_stat_neg, bucket_length, slide_steps);

  AddDataKernel<<<(batch_size + PADDLE_CUDA_NUM_THREADS - 1) /
                      PADDLE_CUDA_NUM_THREADS,
                  PADDLE_CUDA_NUM_THREADS,
                  0,
                  dev_ctx.stream()>>>(label_data,
                                      inference_data,
                                      inference_width,
                                      num_thresholds,
                                      origin_stat_pos,
                                      origin_stat_neg,
                                      batch_size,
                                      slide_steps);
  if (!is_fake_data) {
    UpdateSumDataKernel<<<(bucket_length + PADDLE_CUDA_NUM_THREADS - 1) /
                              PADDLE_CUDA_NUM_THREADS,
                          PADDLE_CUDA_NUM_THREADS,
                          0,
                          dev_ctx.stream()>>>(
        origin_stat_pos, origin_stat_neg, bucket_length, slide_steps);
  }
}

template <typename T, typename Context>
void AucKernel(const Context &dev_ctx,
               const DenseTensor &input,
               const DenseTensor &label,
               const DenseTensor &stat_pos,
               const DenseTensor &stat_neg,
               const paddle::optional<DenseTensor> &ins_tag_weight,
               const std::string &curve,
               int num_thresholds,
               int slide_steps,
               DenseTensor *auc,
               DenseTensor *stat_pos_out,
               DenseTensor *stat_neg_out) {
  // Only use output var for now, make sure it's persistable and
  // not cleaned up for each batch.
  auto *origin_stat_pos = dev_ctx.template Alloc<int64_t>(stat_pos_out);
  auto *origin_stat_neg = dev_ctx.template Alloc<int64_t>(stat_neg_out);
  auto *auc_value = dev_ctx.template Alloc<double>(auc);

  auto *stat_pos_in_tensor = &stat_pos;
  auto *stat_neg_in_tensor = &stat_neg;
  auto *pos_in_data = stat_pos.data<int64_t>();
  auto *neg_in_data = stat_neg.data<int64_t>();
  bool is_fake_data = false;
  if (ins_tag_weight.get_ptr() != nullptr) {
    const auto *ins_tag_weight_data = ins_tag_weight->data<float>();
    if (ins_tag_weight_data[0] == 0) {
      is_fake_data = true;
    }
  }

#ifdef PADDLE_WITH_CUDA
  if (stat_pos_in_tensor != stat_pos_out) {
    hipMemcpy(
        origin_stat_pos,
        pos_in_data,
        ((1 + slide_steps) * (num_thresholds + 1) + (slide_steps > 0 ? 1 : 0)) *
            sizeof(int64_t),
        hipMemcpyDeviceToDevice);
  }
  if (stat_neg_in_tensor != stat_neg_out) {
    hipMemcpy(
        origin_stat_neg,
        neg_in_data,
        ((1 + slide_steps) * (num_thresholds + 1) + (slide_steps > 0 ? 1 : 0)) *
            sizeof(int64_t),
        hipMemcpyDeviceToDevice);
  }
#else
  if (stat_pos_in_tensor != stat_pos_out) {
    hipMemcpy(
        origin_stat_pos,
        pos_in_data,
        ((1 + slide_steps) * (num_thresholds + 1) + (slide_steps > 0 ? 1 : 0)) *
            sizeof(int64_t),
        hipMemcpyDeviceToDevice);
  }
  if (stat_neg_in_tensor != stat_neg_out) {
    hipMemcpy(
        origin_stat_neg,
        neg_in_data,
        ((1 + slide_steps) * (num_thresholds + 1) + (slide_steps > 0 ? 1 : 0)) *
            sizeof(int64_t),
        hipMemcpyDeviceToDevice);
  }
#endif

  // when calculate global_auc && is fake data, just do nothing
  if (slide_steps == 0 && is_fake_data) {
    return;
  }

  statAuc<T, Context>(dev_ctx,
                      label,
                      input,
                      num_thresholds,
                      slide_steps,
                      origin_stat_pos,
                      origin_stat_neg,
                      is_fake_data);
  int sum_offset = slide_steps * (num_thresholds + 1);
  CalcAucKernel<<<1, 1, 0, dev_ctx.stream()>>>(origin_stat_pos + sum_offset,
                                               origin_stat_neg + sum_offset,
                                               num_thresholds,
                                               auc_value,
                                               slide_steps > 0);
}

}  // namespace phi

PD_REGISTER_KERNEL(auc, GPU, ALL_LAYOUT, phi::AucKernel, float) {}
