#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/dist_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/elementwise_subtract_kernel.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"

namespace phi {

#define FULL_MASK 0xffffffff

__device__ __forceinline__ float inline_abs(float x) { return abs(x); }
__device__ __forceinline__ double inline_abs(double x) { return abs(x); }
__device__ __forceinline__ float inline_pow(float base, float exponent) {
  return pow(base, exponent);
}
__device__ __forceinline__ double inline_pow(double base, double exponent) {
  return pow(base, exponent);
}

template <typename T>
__global__ void deviceReduceSumZero(const T* x, T* out, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += static_cast<T>(static_cast<double>(x[i]) != 0);
  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumZeroWithSubstract(const T* x,
                                                 const T* y,
                                                 T* out,
                                                 int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val +=
        inline_abs(static_cast<T>(static_cast<double>(x[i] - y[i]) != 0));
  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumZeroFinal(const T* x, T* out, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += x[i];
  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceMax(const T* x, T* out, int64_t N) {
  T max_val = -1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    max_val = max(max_val, inline_abs(x[i]));
  __syncthreads();
  max_val = phi::funcs::blockReduceMax<T>(max_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = max_val;
}

template <typename T>
__global__ void deviceReduceMaxWithSubstract(const T* x,
                                             const T* y,
                                             T* out,
                                             int64_t N) {
  T max_val = -1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    max_val = max(max_val, inline_abs(x[i] - y[i]));
  __syncthreads();
  max_val = phi::funcs::blockReduceMax<T>(max_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = max_val;
}

template <typename T>
__global__ void deviceReduceMin(const T* x, T* out, int64_t N) {
  T min_val = 1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    min_val = min(min_val, inline_abs(x[i]));
  __syncthreads();
  min_val = phi::funcs::blockReduceMin(min_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = min_val;
}

template <typename T>
__global__ void deviceReduceMinWithSubstract(const T* x,
                                             const T* y,
                                             T* out,
                                             int64_t N) {
  T min_val = 1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    min_val = min(min_val, inline_abs(x[i] - y[i]));
  __syncthreads();
  min_val = phi::funcs::blockReduceMin(min_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = min_val;
}

template <typename T>
__global__ void deviceReduceSumOrder(const T* x, T* out, T p_order, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += static_cast<T>(inline_pow(inline_abs(x[i]), p_order));
  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumOrderWithSubstract(
    const T* x, const T* y, T* out, T p_order, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += static_cast<T>(inline_pow(inline_abs(x[i] - y[i]), p_order));
  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumOrderFinal(const T* x,
                                          T* out,
                                          T p_order,
                                          int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += x[i];
  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = inline_pow(sum_val, (1 / p_order));
}

template <typename T, typename Context>
void DistKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const DenseTensor& y,
                float p,
                DenseTensor* out) {
  DenseTensor intermediate;
  const T* x_ptr = x.data<T>();
  const T* y_ptr = y.data<T>();
  T* o_ptr = dev_ctx.template Alloc<T>(out);
  auto stream = dev_ctx.stream();

  if (x.dims() == y.dims()) {  // same shape
    auto n = x.numel();
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    if (p == 0) {
      deviceReduceSumZeroWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceSumZeroFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == INFINITY) {
      deviceReduceMaxWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMax<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == -INFINITY) {
      deviceReduceMinWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMin<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else {
      T p_order = static_cast<T>(p);
      deviceReduceSumOrderWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, p_order, n);
      hipStreamSynchronize(stream);
      deviceReduceSumOrderFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, p_order, config.block_per_grid.x);
    }

  } else {
    DenseTensor t = Subtract<T, Context>(dev_ctx, x, y);
    const T* t_ptr = t.data<T>();
    auto n = t.numel();
    auto config = backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    if (p == 0) {
      deviceReduceSumZero<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceSumZeroFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == INFINITY) {
      deviceReduceMax<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMax<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == -INFINITY) {
      deviceReduceMin<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMin<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);
    } else {
      T p_order = static_cast<T>(p);
      deviceReduceSumOrder<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, p_order, n);
      hipStreamSynchronize(stream);
      deviceReduceSumOrderFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, p_order, config.block_per_grid.x);
    }

    hipStreamSynchronize(stream);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(dist, GPU, ALL_LAYOUT, phi::DistKernel, float, double) {}
