#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/dist_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/elementwise_subtract_kernel.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/p_norm_kernel.h"

namespace phi {

#define FULL_MASK 0xffffffff

template <typename T>
__global__ void DeviceReduceSumZeroWithSubtract(const T* x,
                                                const T* y,
                                                T* out,
                                                int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    sum_val += abs(static_cast<T>(static_cast<double>(x[i] - y[i]) != 0));
  }

  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void DeviceReduceSumZeroFinal(const T* x, T* out, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    sum_val += x[i];
  }

  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void DeviceReduceMaxWithSubtract(const T* x,
                                            const T* y,
                                            T* out,
                                            int64_t N) {
  T max_val = -1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    max_val = max(max_val, abs(x[i] - y[i]));
  }

  __syncthreads();
  max_val = phi::funcs::blockReduceMax<T>(max_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = max_val;
}

template <typename T>
__global__ void DeviceReduceMaxFinal(const T* x, T* out, int64_t N) {
  T max_val = -1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    max_val = max(max_val, abs(x[i]));
  }
  __syncthreads();
  max_val = phi::funcs::blockReduceMax<T>(max_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = max_val;
}

template <typename T>
__global__ void DeviceReduceMinWithSubtract(const T* x,
                                            const T* y,
                                            T* out,
                                            int64_t N) {
  T min_val = 1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    min_val = min(min_val, abs(x[i] - y[i]));
  }

  __syncthreads();
  min_val = phi::funcs::blockReduceMin(min_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = min_val;
}

template <typename T>
__global__ void DeviceReduceMinFinal(const T* x, T* out, int64_t N) {
  T min_val = 1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    min_val = min(min_val, abs(x[i]));
  }
  __syncthreads();
  min_val = phi::funcs::blockReduceMin(min_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = min_val;
}

template <typename T>
__global__ void DeviceReduceSumPOrderWithSubtract(
    const T* x, const T* y, T* out, T p_order, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    sum_val += static_cast<T>(pow(abs(x[i] - y[i]), p_order));
  }

  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void DeviceReduceSumPOrderFinal(const T* x,
                                           T* out,
                                           T p_order,
                                           int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    sum_val += x[i];
  }

  __syncthreads();
  sum_val = phi::funcs::blockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) out[blockIdx.x] = pow(sum_val, (1 / p_order));
}

template <typename T, typename Context>
void DistKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const DenseTensor& y,
                float p,
                DenseTensor* out) {
  DenseTensor intermediate;
  const T* x_ptr = x.data<T>();
  const T* y_ptr = y.data<T>();
  T* o_ptr = dev_ctx.template Alloc<T>(out);
  auto stream = dev_ctx.stream();

  if (x.dims() == y.dims()) {  // same shape
    auto n = x.numel();
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    if (p == 0) {
      DeviceReduceSumZeroWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      DeviceReduceSumZeroFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == INFINITY) {
      DeviceReduceMaxWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      DeviceReduceMaxFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == -INFINITY) {
      DeviceReduceMinWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      DeviceReduceMinFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else {
      T p_order = static_cast<T>(p);
      DeviceReduceSumPOrderWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, p_order, n);
      DeviceReduceSumPOrderFinal<T>
          <<<1, config.thread_per_block.x, 0, stream>>>(
              i_ptr, o_ptr, p_order, config.block_per_grid.x);
    }

  } else {
    auto t = Subtract<T, Context>(dev_ctx, x, y);
    PNormKernel<T, Context>(dev_ctx, t, p, -1, 1e-12, false, true, out);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(dist, GPU, ALL_LAYOUT, phi::DistKernel, float, double) {}
