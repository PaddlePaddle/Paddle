#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>

#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/dist_kernel.h"
#include "paddle/phi/kernels/elementwise_subtract_kernel.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/p_norm_kernel.h"

namespace phi {

#define FULL_MASK 0xffffffff

template <typename Tx, typename Ty = Tx>
struct ZeroOrderFunctor {
 public:
  HOSTDEVICE explicit inline ZeroOrderFunctor() {}
  HOSTDEVICE inline Ty operator()(const Tx& x, const Tx& y) const {
    return static_cast<Ty>(x != y);
  }
};

template <typename Tx, typename Ty = Tx>
struct OtherOrderFunctor {
  HOSTDEVICE explicit inline OtherOrderFunctor(const Ty& _p_order)
      : p_order(_p_order) {}

  HOSTDEVICE inline Ty operator()(const Tx& x, const Tx& y) const {
    return static_cast<Ty>(
        pow(abs(static_cast<Ty>(x) - static_cast<Ty>(y)), p_order));
  }

 private:
  Ty p_order;
};

template <typename Tx, typename Ty = Tx>
struct PowFunctor {
  HOSTDEVICE explicit inline PowFunctor(const Ty& _p_order)
      : p_order(_p_order) {}
  HOSTDEVICE inline Tx operator()(const Tx x) const {
    return static_cast<Tx>(pow(static_cast<Ty>(x), p_order));
  }
  Ty p_order;
};

template <typename T, typename Functor>
__global__ void ReduceSumWithSubtract(
    const T* x, const T* y, T* out, int64_t N, Functor func) {
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;
  MT sum_val(0.0);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    sum_val += func(x[i], y[i]);
  }

  __syncthreads();
  sum_val = phi::funcs::BlockReduceSum<MT>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = static_cast<T>(sum_val);
  }
}

template <typename T>
__global__ void ReduceMaxWithSubtract(const T* x,
                                      const T* y,
                                      T* out,
                                      int64_t N) {
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;
  MT max_val = std::numeric_limits<MT>::min();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    max_val = max(max_val, abs(static_cast<MT>(x[i]) - static_cast<MT>(y[i])));
  }

  __syncthreads();
  max_val = phi::funcs::BlockReduceMax<MT>(max_val, FULL_MASK);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = static_cast<T>(max_val);
  }
}

template <typename T>
__global__ void ReduceMinWithSubtract(const T* x,
                                      const T* y,
                                      T* out,
                                      int64_t N) {
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;
  MT min_val = std::numeric_limits<MT>::max();
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    min_val = min(min_val, abs(static_cast<MT>(x[i]) - static_cast<MT>(y[i])));
  }

  __syncthreads();
  min_val = phi::funcs::BlockReduceMin<MT>(min_val, FULL_MASK);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = static_cast<T>(min_val);
  }
}

template <typename T, typename Context>
void DistKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const DenseTensor& y,
                float p,
                DenseTensor* out) {
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;
  DenseTensor intermediate;
  const T* x_ptr = x.data<T>();
  const T* y_ptr = y.data<T>();
  T* o_ptr = dev_ctx.template Alloc<T>(out);
  auto stream = dev_ctx.stream();

  auto xdim = x.dims();
  if (xdim == y.dims()) {  // same shape
    auto n = x.numel();
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    std::vector<int64_t> axis_dims = {static_cast<int64_t>(-1)};
    std::vector<int> reduce_axis =
        funcs::details::GetReduceDim(axis_dims, xdim.size(), true);

    if (p == 0) {
      ReduceSumWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n, ZeroOrderFunctor<T>());
      phi::funcs::ReduceKernel<T, T, kps::AddFunctor, kps::IdentityFunctor<MT>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<MT>(), reduce_axis);
    } else if (p == INFINITY) {
      ReduceMaxWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      phi::funcs::ReduceKernel<T, T, kps::MaxFunctor, kps::IdentityFunctor<T>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<T>(), reduce_axis);

    } else if (p == -INFINITY) {
      ReduceMinWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);

      phi::funcs::ReduceKernel<T, T, kps::MinFunctor, kps::IdentityFunctor<T>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<T>(), reduce_axis);

    } else {
      MT p_order = static_cast<MT>(p);
      ReduceSumWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n, OtherOrderFunctor<T, MT>(p_order));
      phi::funcs::ReduceKernel<T, T, kps::AddFunctor, kps::IdentityFunctor<MT>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<MT>(), reduce_axis);

      const DenseTensor* tmp_norm = out;
      std::vector<const DenseTensor*> ins = {tmp_norm};
      std::vector<DenseTensor*> outs = {out};
      MT p_order_ = static_cast<MT>(static_cast<MT>(1.) / p_order);
      phi::funcs::ElementwiseKernel<T>(
          dev_ctx, ins, &outs, PowFunctor<T, MT>(p_order_));
    }

  } else {
    auto t = Subtract<T, Context>(dev_ctx, x, y);
    PNormKernel<T, Context>(dev_ctx, t, p, -1, 1e-12, false, true, out);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(dist,
                   GPU,
                   ALL_LAYOUT,
                   phi::DistKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
