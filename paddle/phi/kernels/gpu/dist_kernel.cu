#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/dist_kernel.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/kernels/elementwise_subtract_kernel.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/p_norm_kernel.h"


namespace phi {

#define FULL_MASK 0xffffffff

template <typename T>
struct ZeroOrderFunctor {
 public:
  __device__ T operator()(const T& x, const T& y) const {
    return static_cast<T>((x - y) != 0);
  }
};

template <typename T>
struct OtherOrderFunctor {
  explicit OtherOrderFunctor(const T& p_order) : p_order_(p_order) {}
  __device__ T operator()(const T& x, const T& y) const {
    using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
    MPType p_order_mp = static_cast<MPType>(p_order_);
    MPType x_mp = static_cast<MPType>(x);
    MPType y_mp = static_cast<MPType>(y);
    return static_cast<T>(pow(abs(x_mp - y_mp), p_order_mp));
  }

 private:
  T p_order_;
};

template <typename T>
struct PowFunctor {
  explicit PowFunctor(const T& p_order) : p_order_(p_order) {}
  HOSTDEVICE inline T operator()(const T x) const {
    using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
    MPType p_order_mp = static_cast<MPType>(p_order_);
    MPType x_mp = static_cast<MPType>(x);
    return static_cast<T>(pow(x_mp, p_order_mp));
  }
  T p_order_;
};

template <typename T, typename Functor>
__global__ void ReduceSumWithSubtract(
    const T* x, const T* y, T* out, int64_t N, Functor func) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType sum_val = static_cast<MPType>(0);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    sum_val += func(x[i], y[i]);
  }

  __syncthreads();
  sum_val = phi::funcs::BlockReduceSum<T>(sum_val, FULL_MASK);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = sum_val;
  }
}

template <typename T>
__global__ void ReduceMaxWithSubtract(const T* x,
                                      const T* y,
                                      T* out,
                                      int64_t N) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType max_val = static_cast<MPType>(-1e10f);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    max_val = max(max_val, abs(x[i] - y[i]));
  }

  __syncthreads();
  max_val = phi::funcs::BlockReduceMax<T>(max_val, FULL_MASK);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = max_val;
  }
}

template <typename T>
__global__ void ReduceMinWithSubtract(const T* x,
                                      const T* y,
                                      T* out,
                                      int64_t N) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType min_val = static_cast<MPType>(1e10f);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    min_val = min(min_val, abs(x[i] - y[i]));
  }

  __syncthreads();
  min_val = phi::funcs::BlockReduceMin(min_val, FULL_MASK);
  if (threadIdx.x == 0) {
    out[blockIdx.x] = min_val;
  }
}

template <typename T, typename Context>
void DistKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const DenseTensor& y,
                float p,
                DenseTensor* out) {
  DenseTensor intermediate;
  const T* x_ptr = x.data<T>();
  const T* y_ptr = y.data<T>();
  T* o_ptr = dev_ctx.template Alloc<T>(out);
  auto stream = dev_ctx.stream();

  auto xdim = x.dims();
  if (xdim == y.dims()) {  // same shape
    auto n = x.numel();
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    std::vector<int64_t> axis_dims = {static_cast<int64_t>(-1)};
    std::vector<int> reduce_axis =
        funcs::details::GetReduceDim(axis_dims, xdim.size(), true);

    if (p == 0) {
      ReduceSumWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n, ZeroOrderFunctor<T>());
      phi::funcs::ReduceKernel<T, T, kps::AddFunctor, kps::IdentityFunctor<T>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<T>(), reduce_axis);

    } else if (p == INFINITY) {
      ReduceMaxWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      phi::funcs::ReduceKernel<T, T, kps::MaxFunctor, kps::IdentityFunctor<T>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<T>(), reduce_axis);

    } else if (p == -INFINITY) {
      ReduceMinWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);

      phi::funcs::ReduceKernel<T, T, kps::MinFunctor, kps::IdentityFunctor<T>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<T>(), reduce_axis);

    } else {
      T p_order = static_cast<T>(p);
      ReduceSumWithSubtract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n, OtherOrderFunctor<T>(p_order));
      phi::funcs::ReduceKernel<T, T, kps::AddFunctor, kps::IdentityFunctor<T>>(
          dev_ctx, intermediate, out, kps::IdentityFunctor<T>(), reduce_axis);

      const DenseTensor* tmp_norm = out;
      std::vector<const DenseTensor*> ins = {tmp_norm};
      std::vector<DenseTensor*> outs = {out};
      T p_order_ = static_cast<T>(1. / p_order);
      phi::funcs::ElementwiseKernel<T>(
          dev_ctx, ins, &outs, PowFunctor<T>(p_order_));
    }

  } else {
    auto t = Subtract<T, Context>(dev_ctx, x, y);
    PNormKernel<T, Context>(dev_ctx, t, p, -1, 1e-12, false, true, out);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(dist, GPU, ALL_LAYOUT, phi::DistKernel, float, double,phi::dtype::float16 ) {}

