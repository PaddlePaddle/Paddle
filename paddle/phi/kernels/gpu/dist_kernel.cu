#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/dist_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/elementwise_subtract_kernel.h"

#define FULL_MASK 0xffffffff
#define HALF_WARP_SIZE 16
#define WARP_SIZE 32

__device__ __forceinline__ float inline_abs(float x) { return abs(x); }
__device__ __forceinline__ double inline_abs(double x) { return abs(x); }
__device__ __forceinline__ float inline_pow(float base, float exponent) {
  return pow(base, exponent);
}
__device__ __forceinline__ double inline_pow(double base, double exponent) {
  return pow(base, exponent);
}

template <typename T>
__forceinline__ __device__ T warpReduceSum(T val) {
#pragma unroll
  for (int offset = HALF_WARP_SIZE; offset > 0; offset /= 2) {
#ifdef __HIPCC__
    val += __shfl_down(val, offset);
#else
    val += __shfl_down_sync(FULL_MASK, val, offset);
#endif
  }
  return val;
}

template <typename T>
__forceinline__ __device__ T warpReduceMax(T val) {
#pragma unroll
  for (int offset = HALF_WARP_SIZE; offset > 0; offset /= 2) {
#ifdef __HIPCC__
    val = max(val, __shfl_xor(val, offset));
#else
    val = max(val, __shfl_xor_sync(FULL_MASK, val, offset));
#endif
  }
  return val;
}

template <typename T>
__forceinline__ __device__ T warpReduceMin(T val) {
#pragma unroll
  for (int offset = HALF_WARP_SIZE; offset > 0; offset /= 2) {
#ifdef __HIPCC__
    val = min(val, __shfl_xor(val, offset));
#else
    val = min(val, __shfl_xor_sync(FULL_MASK, val, offset));
#endif
  }
  return val;
}

template <typename T>
__inline__ __device__ T blockReduceSum(T val) {
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;
  val = warpReduceSum<T>(val);
  __syncthreads();
  if (lane == 0) shared[wid] = val;
  __syncthreads();
  int block_span = blockDim.x >> 5;
  val = (threadIdx.x < block_span) ? shared[lane] : static_cast<T>(0.0f);
  if (wid == 0) val = warpReduceSum<T>(val);
  return val;
}

template <typename T>
__inline__ __device__ T blockReduceMax(T val) {
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;
  val = warpReduceMax<T>(val);
  __syncthreads();
  if (lane == 0) shared[wid] = val;
  __syncthreads();
  int block_span = blockDim.x >> 5;
  val = (threadIdx.x < block_span) ? shared[lane] : -1e10f;
  if (wid == 0) val = warpReduceMax<T>(val);
  return val;
}

template <typename T>
__inline__ __device__ T blockReduceMin(T val) {
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;
  val = warpReduceMax<T>(val);
  __syncthreads();
  if (lane == 0) shared[wid] = val;
  __syncthreads();
  int block_span = blockDim.x >> 5;
  val = (threadIdx.x < block_span) ? shared[lane] : 1e10f;
  if (wid == 0) val = warpReduceMax<T>(val);
  return val;
}

template <typename T>
__global__ void deviceReduceSumZero(const T* x, T* out, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += static_cast<T>(static_cast<double>(x[i]) != 0);
  __syncthreads();
  sum_val = blockReduceSum(sum_val);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumZeroWithSubstract(const T* x,
                                                 const T* y,
                                                 T* out,
                                                 int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val +=
        inline_abs(static_cast<T>(static_cast<double>(x[i] - y[i]) != 0));
  __syncthreads();
  sum_val = blockReduceSum(sum_val);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumZeroFinal(const T* x, T* out, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += x[i];
  __syncthreads();
  sum_val = blockReduceSum(sum_val);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceMax(const T* x, T* out, int64_t N) {
  T max_val = -1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    max_val = max(max_val, inline_abs(x[i]));
  __syncthreads();
  max_val = blockReduceMax(max_val);
  if (threadIdx.x == 0) out[blockIdx.x] = max_val;
}

template <typename T>
__global__ void deviceReduceMaxWithSubstract(const T* x,
                                             const T* y,
                                             T* out,
                                             int64_t N) {
  T max_val = -1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    max_val = max(max_val, inline_abs(x[i] - y[i]));
  __syncthreads();
  max_val = blockReduceMax(max_val);
  if (threadIdx.x == 0) out[blockIdx.x] = max_val;
}

template <typename T>
__global__ void deviceReduceMin(const T* x, T* out, int64_t N) {
  T min_val = 1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    min_val = min(min_val, inline_abs(x[i]));
  __syncthreads();
  min_val = blockReduceMin(min_val);
  if (threadIdx.x == 0) out[blockIdx.x] = min_val;
}

template <typename T>
__global__ void deviceReduceMinWithSubstract(const T* x,
                                             const T* y,
                                             T* out,
                                             int64_t N) {
  T min_val = 1e10f;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    min_val = min(min_val, inline_abs(x[i] - y[i]));
  __syncthreads();
  min_val = blockReduceMin(min_val);
  if (threadIdx.x == 0) out[blockIdx.x] = min_val;
}

template <typename T>
__global__ void deviceReduceSumOrder(const T* x, T* out, T p_order, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += static_cast<T>(inline_pow(inline_abs(x[i]), p_order));
  __syncthreads();
  sum_val = blockReduceSum(sum_val);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumOrderWithSubstract(
    const T* x, const T* y, T* out, T p_order, int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += static_cast<T>(inline_pow(inline_abs(x[i] - y[i]), p_order));
  __syncthreads();
  sum_val = blockReduceSum(sum_val);
  if (threadIdx.x == 0) out[blockIdx.x] = sum_val;
}

template <typename T>
__global__ void deviceReduceSumOrderFinal(const T* x,
                                          T* out,
                                          T p_order,
                                          int64_t N) {
  T sum_val = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x)
    sum_val += x[i];
  __syncthreads();
  sum_val = blockReduceSum(sum_val);
  if (threadIdx.x == 0) out[blockIdx.x] = inline_pow(sum_val, (1 / p_order));
}

namespace phi {

template <typename T, typename Context>
void DistKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const DenseTensor& y,
                float p,
                DenseTensor* out) {
  DenseTensor intermediate;
  const T* x_ptr = x.data<T>();
  const T* y_ptr = y.data<T>();
  T* o_ptr = dev_ctx.template Alloc<T>(out);
  auto stream = dev_ctx.stream();

  if (x.dims() == y.dims()) {  // same shape
    auto n = x.numel();
    auto config = phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    if (p == 0) {
      deviceReduceSumZeroWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceSumZeroFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == INFINITY) {
      deviceReduceMaxWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMax<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == -INFINITY) {
      deviceReduceMinWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMin<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else {
      T p_order = static_cast<T>(p);
      deviceReduceSumOrderWithSubstract<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              x_ptr, y_ptr, i_ptr, p_order, n);
      hipStreamSynchronize(stream);
      deviceReduceSumOrderFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, p_order, config.block_per_grid.x);
    }

  } else {
    DenseTensor t = Subtract<T, Context>(dev_ctx, x, y);
    const T* t_ptr = t.data<T>();
    auto n = t.numel();
    auto config = backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n);
    intermediate.Resize(phi::make_ddim({config.block_per_grid.x}));
    T* i_ptr = dev_ctx.template Alloc<T>(&intermediate);

    if (p == 0) {
      deviceReduceSumZero<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceSumZeroFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == INFINITY) {
      deviceReduceMax<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMax<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);

    } else if (p == -INFINITY) {
      deviceReduceMin<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, n);
      hipStreamSynchronize(stream);
      deviceReduceMin<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, config.block_per_grid.x);
    } else {
      T p_order = static_cast<T>(p);
      deviceReduceSumOrder<T>
          <<<config.block_per_grid.x, config.thread_per_block.x, 0, stream>>>(
              t_ptr, i_ptr, p_order, n);
      hipStreamSynchronize(stream);
      deviceReduceSumOrderFinal<T><<<1, config.thread_per_block.x, 0, stream>>>(
          i_ptr, o_ptr, p_order, config.block_per_grid.x);
    }

    hipStreamSynchronize(stream);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(dist, GPU, ALL_LAYOUT, phi::DistKernel, float, double) {}
