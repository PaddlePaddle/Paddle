#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/sgd_kernel.h"

#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_helper.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T, typename MT>
__global__ void SGDKernelMT(const T* param,
                            const T* grad,
                            const T* learning_rate,
                            const int num,
                            T* param_out,
                            const MT* master_param,
                            MT* master_param_out) {
  MT lr = static_cast<MT>(learning_rate[0]);
  CUDA_KERNEL_LOOP(i, num) {
    MT p_data = master_param ? master_param[i] : static_cast<MT>(param[i]);
    MT g_data = static_cast<MT>(grad[i]);
    p_data = p_data - lr * g_data;
    param_out[i] = static_cast<T>(p_data);
    if (master_param_out) {
      master_param_out[i] = p_data;
    }
  }
}

template <typename T>
__global__ void SparseSGDFunctorKernel(const T* selected_rows,
                                       const int64_t* rows,
                                       const T* learning_rate,
                                       T* tensor_out,
                                       int64_t row_numel,
                                       int64_t limit) {
  for (int64_t i = blockIdx.x; i < limit; i += gridDim.x) {
    const T* selected_rows_ptr = selected_rows + i * row_numel;
    T* tensor_out_ptr = tensor_out + rows[i] * row_numel;
    for (int64_t index = threadIdx.x; index < row_numel; index += blockDim.x) {
      // Since index in rows of SelectedRows can be duplicate, we have to use
      // Atomic Operation to avoid concurrent write error.
      paddle::platform::CudaAtomicAdd(
          tensor_out_ptr + index,
          -static_cast<T>(1.0) * learning_rate[0] * selected_rows_ptr[index]);
    }
  }
}

template <typename T, typename Context>
void SGDDenseKernel(const Context& dev_ctx,
                    const DenseTensor& param,
                    const DenseTensor& learning_rate,
                    const DenseTensor& grad,
                    const paddle::optional<DenseTensor>& master_param,
                    bool multi_precision,
                    DenseTensor* param_out,
                    DenseTensor* master_param_out) {
  using MPDType = typename paddle::operators::details::MPTypeTrait<T>::Type;
  // do check here
  // if (multi_precision) {
  //   bool has_master =
  //       ctx.HasInput("MasterParam") && ctx.HasOutput("MasterParamOut");

  // }
  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision
          ? master_param_out->mutable_data<MPDType>(dev_ctx.GetPlace())
          : nullptr;

  int block = 512;
  int grid = (param.numel() + block - 1) / block;

  SGDKernelMT<T, MPDType><<<grid, block, 0, dev_ctx.stream()>>>(
      param.data<T>(),
      grad.data<T>(),
      learning_rate.data<T>(),
      param.numel(),
      param_out->mutable_data<T>(dev_ctx.GetPlace()),
      master_in_data,
      master_out_data);
}

template <typename T, typename Context>
void SGDDenseParamSparseGradKernel(
    const Context& dev_ctx,
    const DenseTensor& param,
    const DenseTensor& learning_rate,
    const SelectedRows& grad,
    const paddle::optional<DenseTensor>& master_param,
    bool multi_precision,
    DenseTensor* param_out,
    DenseTensor* master_param_out) {
  using MPDType = typename paddle::operators::details::MPTypeTrait<T>::Type;
  // do some check here
  // if (multi_precision) {
  //   bool has_master =
  //       ctx.HasInput("MasterParam") && ctx.HasOutput("MasterParamOut");

  // }
  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision
          ? master_param_out->mutable_data<MPDType>(dev_ctx.GetPlace())
          : nullptr;

  PADDLE_ENFORCE_EQ(
      &param,
      param_out,
      phi::errors::InvalidArgument(
          "The input tensor Param of SgdOp should be equal with ParamOut "
          "if variable's type is SelectedRows."));

  auto in_height = grad.height();
  auto out_dims = param_out->dims();
  PADDLE_ENFORCE_EQ(in_height,
                    out_dims[0],
                    phi::errors::InvalidArgument(
                        "The input tensor Grad's height of SgdOp should be "
                        "equal with ParamOut's dims. But received Grad's "
                        "height [%s] and ParamOut's dims [%s]",
                        in_height,
                        out_dims[0]));

  auto& in_value = grad.value();
  auto& in_rows = grad.rows();

  int64_t in_row_numel = in_value.numel() / in_rows.size();
  PADDLE_ENFORCE_EQ(in_row_numel,
                    param_out->numel() / in_height,
                    phi::errors::InvalidArgument(
                        "The in_row_numel of SgdOp should be equal with "
                        "param_out's numel / in_height."));

  auto* in_data = in_value.data<T>();
  auto* out_data = param_out->data<T>();

  const int kThreadsPerBlock = 256;
  int thread_x = kThreadsPerBlock;
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  int max_blocks = std::max(max_threads / kThreadsPerBlock, 1);
  paddle::framework::MixVector<int64_t> mixv_in_rows(&in_rows);
  SparseSGDFunctorKernel<<<max_blocks, thread_x, 0, dev_ctx.stream()>>>(
      in_data,
      mixv_in_rows.CUDAData(dev_ctx.GetPlace()),
      learning_rate.data<T>(),
      out_data,
      in_row_numel,
      in_rows.size());
}

template <typename T, typename Context>
void SGDSparseParamSparseGradKernel(
    const Context& dev_ctx,
    const SelectedRows& param,
    const DenseTensor& learning_rate,
    const SelectedRows& grad,
    const paddle::optional<SelectedRows>& master_param,
    bool multi_precision,
    SelectedRows* param_out,
    SelectedRows* master_param_out) {
  PADDLE_THROW("not impl");
}

}  // namespace phi

PD_REGISTER_KERNEL(sgd,
                   GPU,
                   ALL_LAYOUT,
                   phi::SGDDenseKernel,
                   phi::dtype::float16,
                   float,
                   double) {}

PD_REGISTER_KERNEL(sgd_dense_param_sparse_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SGDDenseParamSparseGradKernel,
                   phi::dtype::float16,
                   float,
                   double) {}

PD_REGISTER_KERNEL(sgd_sparse_param_sparse_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SGDSparseParamSparseGradKernel,
                   phi::dtype::float16,
                   float,
                   double) {}
