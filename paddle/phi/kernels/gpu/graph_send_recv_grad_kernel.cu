// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/graph_send_recv_grad_kernel.h"

#include <algorithm>
#include <vector>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/hostdevice.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/gpu/graph_send_recv_funcs.h"

namespace phi {

template <typename Context, typename T, typename IndexT>
void GraphSendRecvGradOpCUDAKernelLaunchHelper(
    const Context& ctx,
    const DenseTensor& out_grad,
    const DenseTensor& x,
    const DenseTensor& src_index,
    const DenseTensor& dst_index,
    const std::string& reduce_op,
    DenseTensor* x_grad,
    const DenseTensor* dst_count = nullptr,
    const DenseTensor* out = nullptr) {
  const int& index_size = dst_index.dims()[0];

  ctx.template Alloc<T>(x_grad);
  T* p_output = x_grad->data<T>();

  const auto& src_dims = x.dims();
  int64_t memset_size = 1;
  for (int i = 0; i < src_dims.size(); ++i) {
    memset_size *= src_dims[i];
  }
  const size_t& memset_bytes = memset_size * sizeof(T);

#ifdef PADDLE_WITH_HIP
  hipMemset(p_output, 0, memset_bytes);
#else
  hipMemset(p_output, 0, memset_bytes);
#endif

  if (index_size == 0) return;

  int64_t slice_size = 1;
  for (int i = 1; i < src_dims.size(); ++i) {
    slice_size *= src_dims[i];
  }
  const T* p_src = out_grad.data<T>();
  const IndexT* s_index = src_index.data<IndexT>();
  const IndexT* d_index = dst_index.data<IndexT>();

#ifdef PADDLE_WITH_HIP
  int block = 256;
#else
  int block = 1024;
#endif
  int64_t n = slice_size * index_size;
  int64_t max_grid_dimx = ctx.GetCUDAMaxGridDimSize()[0];
  int64_t grid_tmp = (n + block - 1) / block;
  int64_t grid = grid_tmp < max_grid_dimx ? grid_tmp : max_grid_dimx;
  int64_t input_size = src_dims[0];
  if (reduce_op == "SUM") {
    GraphSendRecvSumCUDAFunctor<T, IndexT> functor;
    GraphSendRecvCUDAKernel<T, IndexT, GraphSendRecvSumCUDAFunctor<T, IndexT>>
        <<<grid, block, 0, ctx.stream()>>>(
            p_src, d_index, s_index, p_output, index_size, slice_size, functor);
  } else if (reduce_op == "MEAN") {
    const int32_t* s_count = dst_count->data<int32_t>();
    ManipulateMeanGradCUDAKernel<T, IndexT><<<grid, block, 0, ctx.stream()>>>(
        p_src, d_index, s_index, p_output, index_size, slice_size, s_count);
  } else if (reduce_op == "MAX" || reduce_op == "MIN") {
    const T* ptr_input = x.data<T>();
    const T* ptr_output = out->data<T>();
    ManipulateMinMaxGradCUDAKernel<T, IndexT>
        <<<grid, block, 0, ctx.stream()>>>(p_src,
                                           d_index,
                                           s_index,
                                           p_output,
                                           index_size,
                                           slice_size,
                                           ptr_input,
                                           ptr_output);
  }
}

template <typename T, typename Context>
void GraphSendRecvGradKernel(const Context& ctx,
                             const DenseTensor& x,
                             const DenseTensor& src_index,
                             const DenseTensor& dst_index,
                             const paddle::optional<DenseTensor>& out,
                             const paddle::optional<DenseTensor>& dst_count,
                             const DenseTensor& out_grad,
                             const std::string& reduce_op,
                             DenseTensor* x_grad) {
  auto index_type = src_index.dtype();
  if (index_type == phi::DataType::INT32) {
    GraphSendRecvGradOpCUDAKernelLaunchHelper<Context, T, int32_t>(
        ctx,
        out_grad,
        x,
        src_index,
        dst_index,
        reduce_op,
        x_grad,
        dst_count.get_ptr(),
        out.get_ptr());
  } else if (index_type == phi::DataType::INT64) {
    GraphSendRecvGradOpCUDAKernelLaunchHelper<Context, T, int64_t>(
        ctx,
        out_grad,
        x,
        src_index,
        dst_index,
        reduce_op,
        x_grad,
        dst_count.get_ptr(),
        out.get_ptr());
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(graph_send_recv_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::GraphSendRecvGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
