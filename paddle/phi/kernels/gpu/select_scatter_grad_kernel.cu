#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/select_scatter_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/place.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/core/utils/data_type.h"

namespace phi {

template <typename T>
__global__ void SelectScatterSrcGradGPUKernel(T* src_grad_data,
                                              int index,
                                              int64_t select_index_size,
                                              int64_t outer_dim_size,
                                              int64_t numel) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numel) return;
  int64_t i, j;
  i = tid / outer_dim_size;
  j = tid % outer_dim_size;
  int64_t src_offset =
      i * select_index_size * outer_dim_size + index * outer_dim_size + j;
  src_grad_data[src_offset] = 0;
}

template <typename T>
__global__ void SelectScatterValuesGradGPUKernel(T* values_grad_data,
                                                 const T* out_grad_data,
                                                 int index,
                                                 int64_t select_index_size,
                                                 int64_t outer_dim_size,
                                                 int64_t numel) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= numel) return;
  int64_t i, j;
  i = tid / outer_dim_size;
  j = tid % outer_dim_size;
  int64_t src_offset =
      i * select_index_size * outer_dim_size + index * outer_dim_size + j;
  int64_t values_offset = i * outer_dim_size + j;
  values_grad_data[values_offset] = out_grad_data[src_offset];
}

template <typename T, typename Context>
void SelectScatterGradKernel(const Context& dev_ctx,
                             const DenseTensor& src,
                             const DenseTensor& values,
                             const DenseTensor& out_grad,
                             int axis,
                             int index,
                             DenseTensor* src_grad,
                             DenseTensor* value_grad) {
  PADDLE_ENFORCE_EQ(dev_ctx.GetPlace().GetType() == phi::AllocationType::GPU,
                    true,
                    errors::PreconditionNotMet(
                        "SelectScatterGradOpCUDAKernel only runs on GPU."));

  if (!src_grad && !value_grad) return;
  auto* out_grad_data = out_grad.data<T>();
  auto src_dims = out_grad.dims();
  int64_t select_index_size = src_dims[axis];
  int64_t inner_dim_size = 1;
  int64_t outer_dim_size = 1;
  for (int i = 0; i < axis; i++) {
    inner_dim_size *= src_dims[i];
  }

  for (int i = axis + 1; i < src_dims.size(); i++) {
    outer_dim_size *= src_dims[i];
  }
  int block = 512;
  int64_t n = inner_dim_size * outer_dim_size;
  int64_t grid = (n + block - 1) / block;
  auto stream = reinterpret_cast<const phi::GPUContext&>(dev_ctx).stream();
  if (src_grad) {
    phi::Copy(dev_ctx, out_grad, dev_ctx.GetPlace(), false, src_grad);
    auto* src_grad_data = src_grad->data<T>();
    SelectScatterSrcGradGPUKernel<T><<<grid, block, 0, stream>>>(
        src_grad_data, index, select_index_size, outer_dim_size, n);
  }
  if (value_grad) {
    value_grad->Resize(values.dims());
    dev_ctx.template Alloc<T>(value_grad);
    auto* value_grad_data = value_grad->data<T>();
    SelectScatterValuesGradGPUKernel<T>
        <<<grid, block, 0, stream>>>(value_grad_data,
                                     out_grad_data,
                                     index,
                                     select_index_size,
                                     outer_dim_size,
                                     n);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(select_scatter_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SelectScatterGradKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
