#include "hip/hip_runtime.h"


// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cpu/reduce.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/funcs/reduce_functor.h"
#include "paddle/phi/kernels/impl/dirichlet_kernel_impl.h"

#ifdef PADDLE_WITH_CUDA
#include <hiprand/hiprand_kernel.h>
#endif
#ifdef PADDLE_WITH_HIP
#include <hiprand_kernel.h>
#endif

#if defined(PADDLE_WITH_CUDA)
using COMPAT_RANDSTATEPHILOX4_32_10_T = hiprandStatePhilox4_32_10_t;
#define COMPAT_RAND_INIT hiprand_init
#define COMPAT_RAND_UNIFORM hiprand_uniform
#define COMPAT_RAND_NORMAL hiprand_normal
#elif defined(PADDLE_WITH_HIP)
using COMPAT_RANDSTATEPHILOX4_32_10_T = hiprandStatePhilox4_32_10_t;
#define COMPAT_RAND_INIT hiprand_init
#define COMPAT_RAND_UNIFORM hiprand_uniform
#define COMPAT_RAND_NORMAL hiprand_normal
#endif

namespace phi {
template <typename T>
struct GammaCUDAFunctor {
  GammaCUDAFunctor(const T* alpha, T* gamma, uint64_t seed, uint64_t offset)
      : alpha_(alpha), gamma_(gamma), seed_(seed), offset_(offset) {}

  DEVICE void operator()(int64_t index) {
    // hiprand initialization
    COMPAT_RANDSTATEPHILOX4_32_10_T state;
    COMPAT_RAND_INIT(
        /*seed=*/seed_, /*subsequence=*/index, /*offset=*/offset_, &state);

    // sample
    auto uniform_lambda = [&state]() { return COMPAT_RAND_UNIFORM(&state); };
    BaseSampler<T, decltype(uniform_lambda)> standard_uniform(uniform_lambda);
    auto normal_lambda = [&state]() { return COMPAT_RAND_NORMAL(&state); };
    BaseSampler<T, decltype(normal_lambda)> standard_normal(normal_lambda);

    auto sample =
        sample_gamma<T, T, decltype(uniform_lambda), decltype(normal_lambda)>(
            alpha_[index], standard_uniform, standard_normal);
    gamma_[index] = std::max(std::numeric_limits<T>::min(), sample);
  }

  const T* alpha_;
  T* gamma_;
  const uint64_t seed_;
  const uint64_t offset_;
};

template <typename T>
struct DirichletSampler<GPUContext, T> {
  void operator()(const GPUContext& dev_ctx,
                  const DenseTensor& alpha,
                  DenseTensor* out) {
    auto p_gen = dev_ctx.GetGenerator();
    auto seed_and_offset = p_gen->IncrementOffset(10);  // hard-coded offset
    auto seed = seed_and_offset.first;
    auto offset = seed_and_offset.second;

    // sample from K gamma distributions, where K=alpha.numel()
    DenseTensor gamma_samples;
    gamma_samples.Resize(alpha.dims());
    dev_ctx.template Alloc<T>(&gamma_samples);

    GammaCUDAFunctor<T> gamma_functor(
        alpha.data<T>(), gamma_samples.data<T>(), seed, offset);
    funcs::ForRange<GPUContext> for_range(dev_ctx, out->numel());
    for_range(gamma_functor);

    // normalize them into a simplex, along the last axis
    DenseTensor gamma_sum;
    auto new_shape = gamma_samples.dims();
    new_shape[new_shape.size() - 1] = 1;
    gamma_sum.Resize(new_shape);
    dev_ctx.template Alloc<T>(&gamma_sum);

    ReduceKernelImpl<GPUContext, T, T, funcs::SumFunctor>(
        dev_ctx,
        gamma_samples,
        &gamma_sum,
        {new_shape.size() - 1},
        true,
        false);
    funcs::ElementwiseCompute<funcs::DivideFunctor<T>, T, T>(
        dev_ctx, gamma_samples, gamma_sum, -1, funcs::DivideFunctor<T>(), out);
  }
};
}  // namespace phi

PD_REGISTER_KERNEL(
    dirichlet, GPU, ALL_LAYOUT, phi::Dirichletkernel, float, double) {}
