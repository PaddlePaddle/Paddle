#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"

#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/memory_utils.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/partial_concat_funcs.h"
#include "paddle/phi/kernels/funcs/strided_memcpy.h"

namespace phi {

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))

template <class T>
__global__ void ConcatPartialGradCUDAKernel(T **in,
                                            const T *out,
                                            int64_t all_length,
                                            int64_t in_batch_len,
                                            int64_t start_index,
                                            int64_t out_batch_len,
                                            int64_t part_length) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while (id < all_length) {
    int64_t bs_id = id / out_batch_len;
    int64_t bs_index = id % out_batch_len;
    int64_t var_id = bs_index / part_length;
    int64_t part_index = bs_index % part_length;
    int64_t in_id = start_index + part_index;
    T *tmp = in[var_id];
    tmp[bs_id * in_batch_len + in_id] = out[id];
    id += blockDim.x * gridDim.x;
  }
}

template <typename T, typename Context>
void PartialConcatGradOpCUDAKernel(const Context &dev_ctx,
                                   const std::vector<const DenseTensor *> &x,
                                   const DenseTensor &out_grad,
                                   int start_index,
                                   int length,
                                   std::vector<DenseTensor *> x_grad) {
  auto ins = x;
  auto outs = x_grad;

  PADDLE_ENFORCE_EQ(ins[0] != nullptr,
                    true,
                    common::errors::InvalidArgument(
                        "The input of partial concat should not be null."));
  // all parameters
  auto batch_size = ins[0]->dims()[0];
  auto in_size = ins[0]->dims()[1];
  // may be negative
  start_index = ComputeStartIndex(start_index, in_size);
  auto partial_len = length;
  if (partial_len < 0) partial_len = in_size - start_index;

  auto in_num = ins.size();
  auto grad_batch_len = partial_len * in_num;
  auto all_length = grad_batch_len * batch_size;
  // initialize
  auto &place = *dev_ctx.eigen_device();
  for (size_t i = 0; i < outs.size(); ++i) {
    dev_ctx.template Alloc<T>(outs[i]);
    auto dxt = phi::EigenVector<T>::Flatten(*outs[i]);
    dxt.device(place) = dxt.constant(static_cast<T>(0));
  }

  constexpr size_t theory_sm_threads = 1024;
  auto stream = dev_ctx.stream();
  auto max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  auto sm_count = max_threads / theory_sm_threads;
  size_t tile_size = 0;
  int grids;
  int blocks;
  auto ComputeKernelParameter = [&](size_t length) {
    if (length >= max_threads)
      tile_size = 1024;
    else if (length < max_threads && length > sm_count * 128)
      tile_size = 512;
    else if (length <= sm_count * 128)
      tile_size = 256;
    grids = CEIL_DIV(length, tile_size);
    blocks = tile_size;
  };

  std::vector<const T *> out_data;
  for (size_t i = 0; i < in_num; ++i) {
    out_data.emplace_back(outs[i]->data<T>());
  }
  auto tmp_out_array = phi::memory_utils::Alloc(
      dev_ctx.GetPlace(),
      out_data.size() * sizeof(T *),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

  phi::memory_utils::Copy(dev_ctx.GetPlace(),
                          tmp_out_array->ptr(),
                          phi::CPUPlace(),
                          reinterpret_cast<void *>(out_data.data()),
                          out_data.size() * sizeof(T *),
                          dev_ctx.stream());

  T **out_grad_data = reinterpret_cast<T **>(tmp_out_array->ptr());
  ComputeKernelParameter(all_length);
  ConcatPartialGradCUDAKernel<T>
      <<<grids, blocks, 0, stream>>>(out_grad_data,
                                     out_grad.data<T>(),
                                     all_length,
                                     in_size,
                                     start_index,
                                     grad_batch_len,
                                     partial_len);
}

}  // namespace phi

PD_REGISTER_KERNEL(partial_concat_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::PartialConcatGradOpCUDAKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
