// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifndef PADDLE_WITH_HIP
// HIP not support cusolver

#include "paddle/phi/kernels/matrix_rank_kernel.h"

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/full_kernel.h"
#include "paddle/phi/kernels/matrix_rank_tol_kernel.h"

namespace phi {

template <typename T, typename Context>
void MatrixRankKernel(const Context& dev_ctx,
                      const DenseTensor& x,
                      float tol,
                      bool use_default_tol,
                      bool hermitian,
                      DenseTensor* out) {
  DenseTensor atol_tensor;
  if (use_default_tol) {
    atol_tensor = phi::Full<T, Context>(dev_ctx, {1}, static_cast<T>(0));
  } else {
    atol_tensor = phi::Full<T, Context>(dev_ctx, {1}, static_cast<T>(tol));
  }
  MatrixRankTolKernel<T, Context>(
      dev_ctx, x, atol_tensor, use_default_tol, hermitian, out);
}

}  // namespace phi

PD_REGISTER_KERNEL(matrix_rank,  // cuda_only
                   GPU,
                   ALL_LAYOUT,
                   phi::MatrixRankKernel,
                   float,
                   double) {}

#endif  // not PADDLE_WITH_HIP
