//   Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/elementwise_grad_kernel.h"
#include "paddle/phi/kernels/funcs/elementwise_functor.h"
#include "paddle/phi/kernels/gpu/elementwise_grad.h"
#include "paddle/phi/kernels/impl/elementwise_grad_kernel_impl.h"

namespace phi {

template <typename T, typename Context>
void SubtractGradKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& y,
                        const DenseTensor& dout,
                        int axis,
                        DenseTensor* dx,
                        DenseTensor* dy) {
  // skip out
  auto* out = &dout;
  if (dx != nullptr && dy != nullptr && (dx->dims() == dy->dims())) {
    elementwise_sub_grad<T>(dev_ctx, x, y, *out, dout, dx, dy);
  } else {
    default_elementwise_sub_grad<T>(dev_ctx, x, y, *out, dout, dx, dy, axis);
  }
}

template <typename T, typename Context>
void SubtractDoubleGradKernel(const Context& dev_ctx,
                              const DenseTensor& y,
                              const DenseTensor& dout,
                              const paddle::optional<DenseTensor>& ddx,
                              const paddle::optional<DenseTensor>& ddy,
                              int axis,
                              DenseTensor* ddout) {
  phi::SubtractDoubleGradImpl<T>(dev_ctx, y, ddx, ddy, dout, axis, ddout);
}

}  // namespace phi

PD_REGISTER_KERNEL(subtract_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SubtractGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}

PD_REGISTER_KERNEL(subtract_double_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::SubtractDoubleGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
