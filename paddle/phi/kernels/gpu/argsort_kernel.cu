#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/argsort_kernel.h"

#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"
#include "paddle/phi/kernels/transpose_kernel.h"

#ifdef __HIPCC__
namespace rocprim {
namespace detail {
template <>
struct radix_key_codec_base<phi::dtype::float16>
    : radix_key_codec_integral<phi::dtype::float16, uint16_t> {};
}  // namespace detail
}  // namespace rocprim
#else
// set cub base traits in order to handle float16
namespace cub {
template <>
struct NumericTraits<phi::dtype::float16>
    : BaseTraits<FLOATING_POINT, true, false, uint16_t, phi::dtype::float16> {};
}  // namespace cub
#endif

namespace phi {

// Iter for move to next row
struct SegmentOffsetIter {
  EIGEN_DEVICE_FUNC
  explicit SegmentOffsetIter(int num_cols) : num_cols_(num_cols) {}

  EIGEN_DEVICE_FUNC EIGEN_STRONG_INLINE int operator()(int idx) const {
    return idx * num_cols_;
  }

  int num_cols_;
};

template <typename T>
static __global__ void FillIndex(T* indices, T num_rows, T num_cols) {
  int col_id = threadIdx.x;
  int row_id = blockIdx.x;

  for (T j = row_id; j < num_rows; j += gridDim.x) {
    for (T i = col_id; i < num_cols; i += blockDim.x) {
      indices[j * num_cols + i] = i;
    }
  }
}

// Sort by flag descending, True: descending. False: Ascending.
// Default is false.
template <typename T, typename IndType>
void ArgFullSort(const phi::GPUContext& ctx,
                 const DenseTensor* input,
                 DenseTensor* output,
                 DenseTensor* indices,
                 const IndType num_rows,
                 const IndType num_cols,
                 const bool descending) {
  auto cu_stream = ctx.stream();
  DenseTensor input_indices;
  const std::vector<IndType> dims = {num_rows, num_cols};
  auto dim = phi::make_ddim(dims);
  input_indices.Resize(dim);
  ctx.template Alloc<IndType>(&input_indices);
  size_t temp_storage_bytes = -1;

  auto ComputeBlockSize = [](IndType col) {
    if (col > 512)
      return 1024;
    else if (col > 256 && col <= 512)
      return 512;
    else if (col > 128 && col <= 256)
      return 256;
    else if (col > 64 && col <= 128)
      return 128;
    else
      return 64;
  };

  int block_size = ComputeBlockSize(num_cols);
  int maxGridDimX = ctx.GetCUDAMaxGridDimSize()[0];
  // actually, int num_rows < max_grid_size
  int grid_size = num_rows < maxGridDimX ? num_rows : maxGridDimX;
  // Init a index array
  FillIndex<<<grid_size, block_size, 0, cu_stream>>>(
      input_indices.data<IndType>(), num_rows, num_cols);

  T* sorted_out_ptr;
  IndType* sorted_indices_ptr;
  const T* inp = input->data<T>();
  T* out = ctx.template Alloc<T>(output);
  IndType* ind = ctx.template Alloc<IndType>(indices);
  sorted_out_ptr = out;
  sorted_indices_ptr = ind;

  // create iter for counting input
  hipcub::CountingInputIterator<IndType> counting_iter(0);
  // segment_offset is used for move to next row
  hipcub::TransformInputIterator<IndType,
                              SegmentOffsetIter,
                              hipcub::CountingInputIterator<IndType>>
      segment_offsets_t(counting_iter, SegmentOffsetIter(num_cols));

  gpuError_t err;
  if (descending) {
    err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        nullptr,
        temp_storage_bytes,
        inp,
        sorted_out_ptr,
        input_indices.data<IndType>(),
        sorted_indices_ptr,
        num_cols * num_rows,
        num_rows,
        segment_offsets_t,
        segment_offsets_t + 1,
        0,
        sizeof(T) * 8,
        cu_stream);
  } else {
    err =
        hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr,
                                                 temp_storage_bytes,
                                                 inp,
                                                 sorted_out_ptr,
                                                 input_indices.data<IndType>(),
                                                 sorted_indices_ptr,
                                                 num_cols * num_rows,
                                                 num_rows,
                                                 segment_offsets_t,
                                                 segment_offsets_t + 1,
                                                 0,
                                                 sizeof(T) * 8,
                                                 cu_stream);
  }
  PADDLE_ENFORCE_GPU_SUCCESS(err);

  DenseTensor temp_storage;
  int64_t temp_size = temp_storage_bytes;
  temp_storage.Resize({temp_size});
  ctx.template Alloc<uint8_t>(&temp_storage);

  if (descending) {
    err = hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        temp_storage.data<uint8_t>(),
        temp_storage_bytes,
        inp,
        sorted_out_ptr,
        input_indices.data<IndType>(),
        sorted_indices_ptr,
        num_cols * num_rows,
        num_rows,
        segment_offsets_t,
        segment_offsets_t + 1,
        0,
        sizeof(T) * 8,
        cu_stream);
  } else {
    err =
        hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage.data<uint8_t>(),
                                                 temp_storage_bytes,
                                                 inp,
                                                 sorted_out_ptr,
                                                 input_indices.data<IndType>(),
                                                 sorted_indices_ptr,
                                                 num_cols * num_rows,
                                                 num_rows,
                                                 segment_offsets_t,
                                                 segment_offsets_t + 1,
                                                 0,
                                                 sizeof(T) * 8,
                                                 cu_stream);
  }

  PADDLE_ENFORCE_GPU_SUCCESS(err);
}

template <typename T, typename Context>
void ArgsortKernel(const Context& dev_ctx,
                   const DenseTensor& input,
                   int axis,
                   bool descending,
                   DenseTensor* output,
                   DenseTensor* indices) {
  auto in_dims = input.dims();
  axis = (axis < 0) ? (in_dims.size() + axis) : axis;
  const T* in_data = input.data<T>();
  auto size = input.numel();
  T* out_data = dev_ctx.template Alloc<T>(output);
  int64_t* ids_data = dev_ctx.template Alloc<int64_t>(indices);

  // Use thrust for parallel acceleration when the input size is equal to the
  // length of the ‘axis’ dimension.
  // Compared to the following 'Special case for full sort', ascending sort is
  // 34 times faster and descending sort is 31 times faster.
  if (size == in_dims[axis]) {
    thrust::sequence(thrust::device, ids_data, ids_data + size);
    thrust::copy(thrust::device, in_data, in_data + size, out_data);
    thrust::sort_by_key(thrust::device, out_data, out_data + size, ids_data);
    if (descending) {
      thrust::reverse(thrust::device, out_data, out_data + size);
      thrust::reverse(thrust::device, ids_data, ids_data + size);
    }
    return;
  }

  // Special case for full sort, speedup ~190x.
  if (axis == -1 || axis + 1 == in_dims.size()) {
    const int64_t input_height =
        phi::product(phi::slice_ddim(in_dims, 0, in_dims.size() - 1));
    const int64_t input_width = in_dims[in_dims.size() - 1];
    ArgFullSort<T, int64_t>(dev_ctx,
                            &input,
                            output,
                            indices,
                            input_height,
                            input_width,
                            descending);
  } else {
    // if not full sort, do transpose first
    std::vector<int> trans;
    for (int i = 0; i < axis; i++) {
      trans.push_back(i);
    }
    trans.push_back(in_dims.size() - 1);
    for (int i = axis + 1; i < in_dims.size() - 1; i++) {
      trans.push_back(i);
    }
    trans.push_back(axis);
    phi::DDim trans_dims(in_dims);
    for (int i = 0; i < trans.size(); i++) {
      trans_dims[i] = in_dims[trans[i]];
    }

    DenseTensor trans_inp;
    trans_inp.Resize(trans_dims);
    T* trans_inp_data = dev_ctx.template Alloc<T>(&trans_inp);
    // Do transpose
    TransposeKernel<T, Context>(dev_ctx, input, trans, &trans_inp);

    const int64_t input_height =
        phi::product(phi::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
    const int64_t input_width = trans_dims[trans_dims.size() - 1];

    DenseTensor tmp_out;
    tmp_out.Resize(trans_dims);
    dev_ctx.template Alloc<T>(&tmp_out);

    DenseTensor tmp_indices;
    // temp indices for sorting
    tmp_indices.Resize(trans_dims);
    dev_ctx.template Alloc<int64_t>(&tmp_indices);
    dev_ctx.template Alloc<int64_t>(indices);

    ArgFullSort<T, int64_t>(dev_ctx,
                            &trans_inp,
                            &tmp_out,
                            &tmp_indices,
                            input_height,
                            input_width,
                            descending);

    TransposeKernel<int64_t, Context>(dev_ctx, tmp_indices, trans, indices);
    // transpose back
    TransposeKernel<T, Context>(dev_ctx, tmp_out, trans, output);
    return;
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(argsort,
                   GPU,
                   ALL_LAYOUT,
                   phi::ArgsortKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
