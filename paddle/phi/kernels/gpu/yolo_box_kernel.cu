#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/yolo_box_kernel.h"

#include "paddle/fluid/memory/malloc.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/yolo_box_util.h"

namespace phi {

template <typename T>
__global__ void KeYoloBoxFw(const T* input,
                            const int* imgsize,
                            T* boxes,
                            T* scores,
                            const float conf_thresh,
                            const int* anchors,
                            const int n,
                            const int h,
                            const int w,
                            const int an_num,
                            const int class_num,
                            const int box_num,
                            int input_size_h,
                            int input_size_w,
                            bool clip_bbox,
                            const float scale,
                            const float bias,
                            bool iou_aware,
                            const float iou_aware_factor) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  T box[4];
  for (; tid < n * box_num; tid += stride) {
    int grid_num = h * w;
    int i = tid / box_num;
    int j = (tid % box_num) / grid_num;
    int k = (tid % grid_num) / w;
    int l = tid % w;

    int an_stride = (5 + class_num) * grid_num;
    int img_height = imgsize[2 * i];
    int img_width = imgsize[2 * i + 1];

    int obj_idx = funcs::GetEntryIndex(
        i, j, k * w + l, an_num, an_stride, grid_num, 4, iou_aware);
    T conf = funcs::sigmoid<T>(input[obj_idx]);
    if (iou_aware) {
      int iou_idx =
          funcs::GetIoUIndex(i, j, k * w + l, an_num, an_stride, grid_num);
      T iou = funcs::sigmoid<T>(input[iou_idx]);
      conf = pow(conf, static_cast<T>(1. - iou_aware_factor)) *
             pow(iou, static_cast<T>(iou_aware_factor));
    }
    if (conf < conf_thresh) {
      continue;
    }

    int box_idx = funcs::GetEntryIndex(
        i, j, k * w + l, an_num, an_stride, grid_num, 0, iou_aware);
    funcs::GetYoloBox<T>(box,
                         input,
                         anchors,
                         l,
                         k,
                         j,
                         h,
                         w,
                         input_size_h,
                         input_size_w,
                         box_idx,
                         grid_num,
                         img_height,
                         img_width,
                         scale,
                         bias);
    box_idx = (i * box_num + j * grid_num + k * w + l) * 4;
    funcs::CalcDetectionBox<T>(
        boxes, box, box_idx, img_height, img_width, clip_bbox);

    int label_idx = funcs::GetEntryIndex(
        i, j, k * w + l, an_num, an_stride, grid_num, 5, iou_aware);
    int score_idx = (i * box_num + j * grid_num + k * w + l) * class_num;
    funcs::CalcLabelScore<T>(
        scores, input, label_idx, score_idx, class_num, conf, grid_num);
  }
}

template <typename T, typename Context>
void YoloBoxKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& img_size,
                   const std::vector<int>& anchors,
                   int class_num,
                   float conf_thresh,
                   int downsample_ratio,
                   bool clip_bbox,
                   float scale_x_y,
                   bool iou_aware,
                   float iou_aware_factor,
                   DenseTensor* boxes,
                   DenseTensor* scores) {
  auto* input = &x;
  float scale = scale_x_y;
  float bias = -0.5 * (scale - 1.);

  const int n = input->dims()[0];
  const int h = input->dims()[2];
  const int w = input->dims()[3];
  const int box_num = boxes->dims()[1];
  const int an_num = anchors.size() / 2;
  int input_size_h = downsample_ratio * h;
  int input_size_w = downsample_ratio * w;

  int bytes = sizeof(int) * anchors.size();
  DenseTensor tmp_anchors;
  tmp_anchors.Resize(phi::make_dim(anchors.size()));
  int* anchors_data = dev_ctx.template Alloc<int>(&tmp_anchors);
  const auto gplace = dev_ctx.GetPlace();
  const auto cplace = phi::CPUPlace();
  paddle::memory::Copy(
      gplace, anchors_data, cplace, anchors.data(), bytes, dev_ctx.stream());

  const T* input_data = input->data<T>();
  const int* imgsize_data = img_size.data<int>();
  T* boxes_data = boxes->mutable_data<T>({n, box_num, 4}, dev_ctx.GetPlace());
  T* scores_data =
      scores->mutable_data<T>({n, box_num, class_num}, dev_ctx.GetPlace());
  phi::funcs::SetConstant<phi::GPUContext, T> set_zero;
  set_zero(dev_ctx, boxes, static_cast<T>(0));
  set_zero(dev_ctx, scores, static_cast<T>(0));
  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, n * box_num);

  dim3 thread_num = config.thread_per_block;
#ifdef WITH_NV_JETSON
  if (config.compute_capability == 53 || config.compute_capability == 62) {
    thread_num = 512;
  }
#endif

  KeYoloBoxFw<T><<<config.block_per_grid, thread_num, 0, dev_ctx.stream()>>>(
      input_data,
      imgsize_data,
      boxes_data,
      scores_data,
      conf_thresh,
      anchors_data,
      n,
      h,
      w,
      an_num,
      class_num,
      box_num,
      input_size_h,
      input_size_w,
      clip_bbox,
      scale,
      bias,
      iou_aware,
      iou_aware_factor);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    yolo_box, GPU, ALL_LAYOUT, phi::YoloBoxKernel, float, double) {}
