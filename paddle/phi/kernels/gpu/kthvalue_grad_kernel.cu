#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/kthvalue_grad_kernel.h"

#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {
static int getBlockSize(int col) {
  if (col > 512)
    return 1024;
  else if (col > 256 && col <= 512)
    return 512;
  else if (col > 128 && col <= 256)
    return 256;
  else if (col > 64 && col <= 128)
    return 128;
  else
    return 64;
}

template <typename T, typename Context>
void KthvalueGradKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& indices,
                        const DenseTensor& d_out,
                        int k,
                        int axis,
                        bool keepdim,
                        DenseTensor* d_x) {
  const auto& in_dims = x.dims();
  auto out_dims = indices.dims();
  if (axis < 0) axis += in_dims.size();
  T* x_grad_data = dev_ctx.template Alloc<T>(d_x);
  const T* out_grad_data = d_out.data<T>();
  const int64_t* indices_data = indices.data<int64_t>();
  int pre, n, post;
  paddle::operators::GetDims(in_dims, axis, &pre, &n, &post);
  int block_size = getBlockSize(post * k);
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
  int grid_size = std::min(max_blocks, pre);
  paddle::operators::AssignGradWithAxis<T>
      <<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
          out_grad_data, indices_data, x_grad_data, pre, post, n, 1);
}

}  // namespace phi

PD_REGISTER_KERNEL(kthvalue_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::KthvalueGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
