#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/kthvalue_grad_kernel.h"

<<<<<<< HEAD
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/top_k_function_cuda.h"
=======
#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81

namespace phi {
static int getBlockSize(int col) {
  if (col > 512)
    return 1024;
  else if (col > 256 && col <= 512)
    return 512;
  else if (col > 128 && col <= 256)
    return 256;
  else if (col > 64 && col <= 128)
    return 128;
  else
    return 64;
}

template <typename T, typename Context>
void KthvalueGradKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& indices,
                        const DenseTensor& d_out,
                        int k,
                        int axis,
                        bool keepdim,
                        DenseTensor* d_x) {
  const auto& in_dims = x.dims();
  auto out_dims = indices.dims();
<<<<<<< HEAD
  T* x_grad_data = dev_ctx.template Alloc<T>(d_x);
  // For 0D Tensor
  if (in_dims.size() == 0) {
    phi::funcs::set_constant(dev_ctx, d_x, 1.0);
    return;
  }

  if (axis < 0) axis += in_dims.size();

  const T* out_grad_data = d_out.data<T>();
  const int64_t* indices_data = indices.data<int64_t>();
  int pre, n, post;
  phi::funcs::GetDims(in_dims, axis, &pre, &n, &post);
=======
  if (axis < 0) axis += in_dims.size();
  T* x_grad_data = dev_ctx.template Alloc<T>(d_x);
  const T* out_grad_data = d_out.data<T>();
  const int64_t* indices_data = indices.data<int64_t>();
  int pre, n, post;
  paddle::operators::GetDims(in_dims, axis, &pre, &n, &post);
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
  int block_size = getBlockSize(post * k);
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
  int grid_size = std::min(max_blocks, pre);
<<<<<<< HEAD
  phi::funcs::AssignGradWithAxis<T>
=======
  paddle::operators::AssignGradWithAxis<T>
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
      <<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
          out_grad_data, indices_data, x_grad_data, pre, post, n, 1);
}

}  // namespace phi

PD_REGISTER_KERNEL(kthvalue_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::KthvalueGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
