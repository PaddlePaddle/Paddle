/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/scale_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
// See Note [ Why still include the fluid headers? ]
#include "paddle/phi/common/float16.h"

namespace phi {

template <typename InT>
struct ScaleFunctor {
  InT bias;
  InT scale;
  bool bias_after_scale;

  ScaleFunctor(InT scale_data, InT bias_data, bool is_bias_after_sacle)
      : bias(bias_data),
        scale(scale_data),
        bias_after_scale(is_bias_after_sacle) {}

  __device__ __forceinline__ InT operator()(const InT x) const {
    if (bias_after_scale) {
      return scale * x + bias;
    } else {
      return scale * (x + bias);
    }
  }
};

template <typename T, typename Context>
void ScaleKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const Scalar& scale,
                 float bias,
                 bool bias_after_scale,
                 DenseTensor* out) {
  std::vector<const DenseTensor*> inputs;
  std::vector<DenseTensor*> outputs;
  inputs.emplace_back(&x);
  outputs.emplace_back(out);
  dev_ctx.template Alloc<T>(out);
  phi::funcs::ElementwiseKernel<T>(
      dev_ctx,
      inputs,
      &outputs,
      ScaleFunctor<T>(scale.to<T>(), static_cast<T>(bias), bias_after_scale));
}

}  // namespace phi

PD_REGISTER_KERNEL(scale,
                   GPU,
                   ALL_LAYOUT,
                   phi::ScaleKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
