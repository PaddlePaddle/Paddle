// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/split_kernel.h"

#include "paddle/fluid/operators/strided_memcpy.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"

#include "paddle/phi/kernels/gpu/concat_and_split.h"
namespace phi {

template <typename T, typename Context>
void SplitKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const ScalarArray& num_or_sections,
                 const Scalar& axis_scalar,
                 std::vector<DenseTensor*> outs) {
  // need to infershape output
  if (num_or_sections.FromTensor() || axis_scalar.FromTensor()) {
    std::vector<MetaTensor> out_metas;
    out_metas.reserve(outs.size());
    std::vector<MetaTensor*> out_metas_ptr;
    for (size_t i = 0; i < outs.size(); ++i) {
      out_metas.emplace_back(outs[i]);
      out_metas_ptr.push_back(&out_metas.back());
    }
    phi::SplitInferMeta(x, num_or_sections, axis_scalar, out_metas_ptr, true);
  }

  std::vector<const DenseTensor*> shape_refer;
  for (size_t j = 0; j < outs.size(); ++j) {
    dev_ctx.template Alloc<T>(outs[j]);
    shape_refer.emplace_back(outs[j]);
  }

  int axis = axis_scalar.to<int>();
  // Sometimes direct copies will be faster, this maybe need deeply analysis.
  if (axis == 0 && outs.size() < 10) {
    paddle::operators::StridedMemcpyWithAxis0<T>(
        dev_ctx, x, shape_refer, &outs);
  } else {
    SplitImpl<T, Context>(dev_ctx, x, shape_refer, axis, &outs);
  }
}

}  // namespace phi

PT_REGISTER_KERNEL(split,
                   GPU,
                   ALL_LAYOUT,
                   phi::SplitKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   bool,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
