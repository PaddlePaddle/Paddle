#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/fake_quantize_abs_max_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T, typename Context>
void FakeQuantizeAbsMaxKernel(const Context &dev_ctx,
                              const DenseTensor &x,
                              int bit_length,
                              int round_type,
                              DenseTensor *out,
                              DenseTensor *out_scale) {
  T *out_s = dev_ctx.template Alloc<T>(out_scale);
  int bin_cnt = std::pow(2, bit_length - 1) - 1;
  const T *in_data = x.data<T>();
  phi::funcs::FindAbsMaxFunctor<Context, T> find_abs_max_functor;
  find_abs_max_functor(dev_ctx, in_data, x.numel(), out_s);

  phi::funcs::ClipAndFakeQuantFunctor<Context, T> clip_and_fake_quant_functor;
  clip_and_fake_quant_functor(dev_ctx, x, *out_scale, bin_cnt, round_type, out);
}

}  // namespace phi

PD_REGISTER_KERNEL(fake_quantize_abs_max,
                   GPU,
                   ALL_LAYOUT,
                   phi::FakeQuantizeAbsMaxKernel,
                   float,
                   phi::dtype::float16) {}
