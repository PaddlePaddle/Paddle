#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/layout_utils.h"
#include "paddle/fluid/operators/norm_utils.cu.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/flags.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/batch_norm_kernel.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/norm_utils.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/gpu/batch_norm_utils.h"

#ifdef __HIPCC__
#define LAUNCH_BOUNDS(BlockDim) __launch_bounds__(BlockDim)
#else
#define LAUNCH_BOUNDS(BlockDim)
#endif

DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace phi {

template <typename T>
using CudnnDataType = paddle::platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T, phi::DataLayout layout>
static __global__ void BNForwardInference(const T *x,
                                          const BatchNormParamType<T> *mean,
                                          const BatchNormParamType<T> *variance,
                                          const BatchNormParamType<T> *scale,
                                          const BatchNormParamType<T> *bias,
                                          const int C,
                                          const int N,
                                          const int HxW,
                                          const double epsilon,
                                          T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int num = N * C * HxW;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == phi::DataLayout::kNCHW ? i / HxW % C : i % C;
    BatchNormParamType<T> x_sub_mean =
        static_cast<BatchNormParamType<T>>(x[i]) - mean[c];
    BatchNormParamType<T> inv_var = 1 / sqrt(variance[c] + epsilon);
    y[i] = static_cast<T>(scale[c] * x_sub_mean * inv_var + bias[c]);
  }
}

template <typename T>
static __global__ void InverseVariance(const BatchNormParamType<T> *variance,
                                       const double epsilon,
                                       const int C,
                                       BatchNormParamType<T> *inv_variance) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < C) {
    inv_variance[tid] = 1 / sqrt(variance[tid] + epsilon);
  }
}

template <typename T, phi::DataLayout layout>
static __global__ void BN1DForwardInference(
    const T *x,
    const BatchNormParamType<T> *mean,
    const BatchNormParamType<T> *inv_variance,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    const double epsilon,
    T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int num = N * C * HxW;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == phi::DataLayout::kNCHW ? i / HxW % C : i % C;
    BatchNormParamType<T> x_sub_mean =
        static_cast<BatchNormParamType<T>>(x[i]) - mean[c];
    y[i] = static_cast<T>(scale[c] * x_sub_mean * inv_variance[c] + bias[c]);
  }
}

template <typename T, int BlockDim, phi::DataLayout layout>
static __global__ LAUNCH_BOUNDS(BlockDim) void BNForwardTraining(
    const T *x,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    const double epsilon,
    double exponentialAverageFactor,
    T *y,
    BatchNormParamType<T> *mean,
    BatchNormParamType<T> *variance,
    BatchNormParamType<T> *save_mean,
    BatchNormParamType<T> *save_inv_variance) {
  int outer_size = C;
  int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage mean_storage;
  __shared__ typename BlockReduce::TempStorage variance_storeage;
  __shared__ BatchNormParamType<T> mean_val;
  __shared__ BatchNormParamType<T> variance_val;
  __shared__ BatchNormParamType<T> inv_var_val;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> x_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> x_square_sum = static_cast<BatchNormParamType<T>>(0);

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == phi::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      BatchNormParamType<T> x_i = static_cast<BatchNormParamType<T>>(x[index]);
      x_sum += x_i;
      x_square_sum += x_i * x_i;
    }
    x_sum = BlockReduce(mean_storage).Reduce(x_sum, hipcub::Sum());
    x_square_sum =
        BlockReduce(variance_storeage).Reduce(x_square_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      mean_val = x_sum / inner_size;
      variance_val = x_square_sum / inner_size - mean_val * mean_val;
      inv_var_val = 1 / sqrt(variance_val + epsilon);

      if (save_mean && save_inv_variance) {
        save_mean[i] = mean_val;
        save_inv_variance[i] = inv_var_val;
      }
      mean[i] = (1 - exponentialAverageFactor) * mean_val +
                exponentialAverageFactor * mean[i];
      variance[i] = (1 - exponentialAverageFactor) * variance_val +
                    exponentialAverageFactor * variance[i];
    }
    __syncthreads();

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == phi::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      BatchNormParamType<T> x_sub_mean =
          static_cast<BatchNormParamType<T>>(x[index]) - mean_val;
      y[index] = scale[i] * x_sub_mean * inv_var_val + bias[i];
    }
  }
}

template <typename T>
__device__ __forceinline__ void merge_block_vertical(
    BatchNormParamType<T> x_sum,
    BatchNormParamType<T> x_square_sum,
    BatchNormParamType<T> *smem_sum,
    BatchNormParamType<T> *smem_square_sum,
    BatchNormParamType<T> *x_sum_out,
    BatchNormParamType<T> *x_square_sum_out) {
  int tid = threadIdx.x + threadIdx.y * blockDim.x;
#pragma unroll
  for (int offset = blockDim.y / 2; offset > 0; offset >>= 1) {
    if (threadIdx.y < offset * 2) {
      smem_sum[tid] = x_sum;
      smem_square_sum[tid] = x_square_sum;
    }
    __syncthreads();
    if (threadIdx.y < offset) {
      int pair_tid = tid + offset * blockDim.x;
      x_sum += smem_sum[pair_tid];
      x_square_sum += smem_square_sum[pair_tid];
    }
  }
  if (threadIdx.y == 0) {
    *x_sum_out = x_sum;
    *x_square_sum_out = x_square_sum;
  }
}

template <typename T>
__device__ __forceinline__ void merge_block_horizonal(
    BatchNormParamType<T> x_sum,
    BatchNormParamType<T> x_square_sum,
    BatchNormParamType<T> *smem_sum,
    BatchNormParamType<T> *smem_square_sum,
    BatchNormParamType<T> *x_sum_out,
    BatchNormParamType<T> *x_square_sum_out) {
  int tid = threadIdx.x + threadIdx.y * blockDim.x;
#pragma unroll
  for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset * 2) {
      smem_sum[tid] = x_sum;
      smem_square_sum[tid] = x_square_sum;
    }
    __syncthreads();
    if (threadIdx.x < offset) {
      int pair_tid = tid + offset;
      x_sum += smem_sum[pair_tid];
      x_square_sum += smem_square_sum[pair_tid];
    }
  }
  if (threadIdx.x == 0) {
    *x_sum_out = x_sum;
    *x_square_sum_out = x_square_sum;
  }
}

template <typename T, int BlockDim>
static __global__ void BNForwardTraining2DChannelLastCompStat(
    const T *x,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    const double epsilon,
    double exponentialAverageFactor,
    T *y,
    BatchNormParamType<T> *global_mean,
    BatchNormParamType<T> *global_variance,
    BatchNormParamType<T> *save_mean,
    BatchNormParamType<T> *save_inv_variance,
    BatchNormParamType<T> *compute_mean,
    BatchNormParamType<T> *compute_inv_var,
    BatchNormParamType<T> *block_data_ptr,
    int *flag_ptr) {
  int outer_size = C;
  int inner_size = N * HxW;

  __shared__ BatchNormParamType<T> smem_sum[BlockDim];
  __shared__ BatchNormParamType<T> smem_square_sum[BlockDim];

  int outer_loop_stride = gridDim.x * blockDim.x;
  int inner_loop_stride = gridDim.y * blockDim.y;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < outer_size;
       i += outer_loop_stride) {
    BatchNormParamType<T> x_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> x_square_sum = static_cast<BatchNormParamType<T>>(0);

    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < inner_size;
         j += inner_loop_stride) {
      const int index = j * outer_size + i;
      BatchNormParamType<T> x_i = static_cast<BatchNormParamType<T>>(x[index]);
      x_sum += x_i;
      x_square_sum += x_i * x_i;
    }

    // vertical block sum
    merge_block_vertical<T>(x_sum,
                            x_square_sum,
                            &smem_sum[0],
                            &smem_square_sum[0],
                            &x_sum,
                            &x_square_sum);

    if (gridDim.y > 1) {
      volatile BatchNormParamType<T> *staging_sum = block_data_ptr;
      volatile BatchNormParamType<T> *staging_square_sum =
          &block_data_ptr[C * gridDim.y];
      // write block data to global memory
      if (threadIdx.y == 0) {
        staging_sum[i + blockIdx.y * C] = x_sum;
        staging_square_sum[i + blockIdx.y * C] = x_square_sum;
      }

      // make sure write is visible to all blocks
      __threadfence();
      __syncthreads();

      __shared__ bool is_last_block_done;
      // mark block done
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        int old = atomicAdd(&flag_ptr[blockIdx.x], 1);
        is_last_block_done = (old == (gridDim.y - 1));
      }

      __syncthreads();

      if (is_last_block_done) {
        x_sum = static_cast<BatchNormParamType<T>>(0);
        x_square_sum = static_cast<BatchNormParamType<T>>(0);
        // thread sum
        for (int y = threadIdx.y; y < gridDim.y; y += blockDim.y) {
          x_sum += staging_sum[i + y * C];
          x_square_sum += staging_square_sum[i + y * C];
        }

        // vertical block sum
        merge_block_vertical<T>(x_sum,
                                x_square_sum,
                                &smem_sum[0],
                                &smem_square_sum[0],
                                &x_sum,
                                &x_square_sum);

        // final compute
        if (threadIdx.y == 0) {
          BatchNormParamType<T> compute_mean_val = x_sum / inner_size;
          BatchNormParamType<T> variance_val =
              x_square_sum / inner_size - compute_mean_val * compute_mean_val;
          BatchNormParamType<T> compute_inv_var_val =
              1 / sqrt(variance_val + epsilon);

          if (save_mean && save_inv_variance) {
            save_mean[i] = compute_mean_val;
            save_inv_variance[i] = compute_inv_var_val;
          }
          global_mean[i] = (1 - exponentialAverageFactor) * compute_mean_val +
                           exponentialAverageFactor * global_mean[i];
          global_variance[i] = (1 - exponentialAverageFactor) * variance_val +
                               exponentialAverageFactor * global_variance[i];

          compute_mean[i] = compute_mean_val;
          compute_inv_var[i] = compute_inv_var_val;
        }
      }
    } else {
      if (blockIdx.y == 0 && threadIdx.y == 0) {
        BatchNormParamType<T> compute_mean_val = x_sum / inner_size;
        BatchNormParamType<T> variance_val =
            x_square_sum / inner_size - compute_mean_val * compute_mean_val;
        BatchNormParamType<T> compute_inv_var_val =
            1 / sqrt(variance_val + epsilon);

        if (save_mean && save_inv_variance) {
          save_mean[i] = compute_mean_val;
          save_inv_variance[i] = compute_inv_var_val;
        }
        global_mean[i] = (1 - exponentialAverageFactor) * compute_mean_val +
                         exponentialAverageFactor * global_mean[i];
        global_variance[i] = (1 - exponentialAverageFactor) * variance_val +
                             exponentialAverageFactor * global_variance[i];

        compute_mean[i] = compute_mean_val;
        compute_inv_var[i] = compute_inv_var_val;
      }
    }
  }
}

template <typename T>
static __global__ void BNForwardTraining2DChannelLastWriteRes(
    const T *x,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    T *y,
    BatchNormParamType<T> *compute_mean,
    BatchNormParamType<T> *compute_inv_var) {
  int outer_size = C;
  int inner_size = N * HxW;

  int outer_loop_stride = gridDim.x * blockDim.x;
  int inner_loop_stride = gridDim.y * blockDim.y;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < outer_size;
       i += outer_loop_stride) {
    BatchNormParamType<T> mean_val = compute_mean[i];
    BatchNormParamType<T> inv_var_val = compute_inv_var[i];
    BatchNormParamType<T> scale_val = scale[i];
    BatchNormParamType<T> bias_val = bias[i];

    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < inner_size;
         j += inner_loop_stride) {
      const int index = j * outer_size + i;
      BatchNormParamType<T> x_sub_mean =
          static_cast<BatchNormParamType<T>>(x[index]) - mean_val;
      y[index] = scale_val * x_sub_mean * inv_var_val + bias_val;
    }
  }
}

template <typename T, int BlockDim>
static __global__ void BNForwardTraining2DCompStat(
    const T *x,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    const double epsilon,
    double exponentialAverageFactor,
    T *y,
    BatchNormParamType<T> *global_mean,
    BatchNormParamType<T> *global_variance,
    BatchNormParamType<T> *save_mean,
    BatchNormParamType<T> *save_inv_variance,
    BatchNormParamType<T> *compute_mean,
    BatchNormParamType<T> *compute_inv_var,
    BatchNormParamType<T> *block_data_ptr,
    int *flag_ptr) {
  int outer_size = C;
  int inner_size = N * HxW;

  __shared__ BatchNormParamType<T> smem_sum[BlockDim];
  __shared__ BatchNormParamType<T> smem_square_sum[BlockDim];

  int outer_loop_stride = gridDim.y * blockDim.y;
  int inner_loop_stride = gridDim.x * blockDim.x;

  for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < outer_size;
       i += outer_loop_stride) {
    BatchNormParamType<T> x_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> x_square_sum = static_cast<BatchNormParamType<T>>(0);

    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < inner_size;
         j += inner_loop_stride) {
      const int index = (j / HxW * C + i) * HxW + j % HxW;
      BatchNormParamType<T> x_i = static_cast<BatchNormParamType<T>>(x[index]);
      x_sum += x_i;
      x_square_sum += x_i * x_i;
    }

    // horizonal block sum
    merge_block_horizonal<T>(x_sum,
                             x_square_sum,
                             &smem_sum[0],
                             &smem_square_sum[0],
                             &x_sum,
                             &x_square_sum);

    if (gridDim.x > 1) {
      volatile BatchNormParamType<T> *staging_sum = block_data_ptr;
      volatile BatchNormParamType<T> *staging_square_sum =
          &block_data_ptr[C * gridDim.x];
      // write block data to global memory
      if (threadIdx.x == 0) {
        staging_sum[i + blockIdx.x * C] = x_sum;
        staging_square_sum[i + blockIdx.x * C] = x_square_sum;
      }

      // make sure write is visible to all blocks
      __threadfence();
      __syncthreads();

      __shared__ bool is_last_block_done;
      // mark block done
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        int old = atomicAdd(&flag_ptr[blockIdx.y], 1);
        is_last_block_done = (old == (gridDim.x - 1));
      }

      __syncthreads();

      if (is_last_block_done) {
        x_sum = static_cast<BatchNormParamType<T>>(0);
        x_square_sum = static_cast<BatchNormParamType<T>>(0);
        // thread sum
        for (int x = threadIdx.x; x < gridDim.x; x += blockDim.x) {
          x_sum += staging_sum[i + x * C];
          x_square_sum += staging_square_sum[i + x * C];
        }

        // horizonal block sum
        merge_block_horizonal<T>(x_sum,
                                 x_square_sum,
                                 &smem_sum[0],
                                 &smem_square_sum[0],
                                 &x_sum,
                                 &x_square_sum);

        // final compute
        if (threadIdx.x == 0) {
          BatchNormParamType<T> compute_mean_val = x_sum / inner_size;
          BatchNormParamType<T> variance_val =
              x_square_sum / inner_size - compute_mean_val * compute_mean_val;
          BatchNormParamType<T> compute_inv_var_val =
              1 / sqrt(variance_val + epsilon);

          if (save_mean && save_inv_variance) {
            save_mean[i] = compute_mean_val;
            save_inv_variance[i] = compute_inv_var_val;
          }
          global_mean[i] = (1 - exponentialAverageFactor) * compute_mean_val +
                           exponentialAverageFactor * global_mean[i];
          global_variance[i] = (1 - exponentialAverageFactor) * variance_val +
                               exponentialAverageFactor * global_variance[i];

          compute_mean[i] = compute_mean_val;
          compute_inv_var[i] = compute_inv_var_val;
        }
      }
    } else {
      if (blockIdx.x == 0 && threadIdx.x == 0) {
        BatchNormParamType<T> compute_mean_val = x_sum / inner_size;
        BatchNormParamType<T> variance_val =
            x_square_sum / inner_size - compute_mean_val * compute_mean_val;
        BatchNormParamType<T> compute_inv_var_val =
            1 / sqrt(variance_val + epsilon);

        if (save_mean && save_inv_variance) {
          save_mean[i] = compute_mean_val;
          save_inv_variance[i] = compute_inv_var_val;
        }
        global_mean[i] = (1 - exponentialAverageFactor) * compute_mean_val +
                         exponentialAverageFactor * global_mean[i];
        global_variance[i] = (1 - exponentialAverageFactor) * variance_val +
                             exponentialAverageFactor * global_variance[i];

        compute_mean[i] = compute_mean_val;
        compute_inv_var[i] = compute_inv_var_val;
      }
    }
  }
}

template <typename T>
static __global__ void BNForwardTraining2DWriteRes(
    const T *x,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    T *y,
    BatchNormParamType<T> *compute_mean,
    BatchNormParamType<T> *compute_inv_var) {
  int outer_size = C;
  int inner_size = N * HxW;

  int outer_loop_stride = gridDim.y * blockDim.y;
  int inner_loop_stride = gridDim.x * blockDim.x;

  for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < outer_size;
       i += outer_loop_stride) {
    BatchNormParamType<T> mean_val = compute_mean[i];
    BatchNormParamType<T> inv_var_val = compute_inv_var[i];
    BatchNormParamType<T> scale_val = scale[i];
    BatchNormParamType<T> bias_val = bias[i];

    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < inner_size;
         j += inner_loop_stride) {
      const int index = (j / HxW * C + i) * HxW + j % HxW;
      BatchNormParamType<T> x_sub_mean =
          static_cast<BatchNormParamType<T>>(x[index]) - mean_val;
      y[index] = scale_val * x_sub_mean * inv_var_val + bias_val;
    }
  }
}

template <typename T, typename Context>
void BatchNormKernel(const Context &ctx,
                     const DenseTensor &x,
                     const DenseTensor &scale,
                     const DenseTensor &bias,
                     const DenseTensor &mean,
                     const DenseTensor &variance,
                     float momentum,
                     float epsilon_f,
                     const std::string &data_layout_str,
                     bool is_test,
                     bool use_global_stats,
                     bool trainable_statistics,
                     bool fuse_with_relu,
                     DenseTensor *y,
                     DenseTensor *mean_out,
                     DenseTensor *variance_out,
                     DenseTensor *saved_mean,
                     DenseTensor *saved_variance,
                     DenseTensor *reserve_space) {
  double epsilon = epsilon_f;
  const bool trainable_stats = trainable_statistics;
  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_layout_str);
  bool test_mode = is_test && (!trainable_stats);

  // Get the size for each dimension.
  // NCHW [batch_size, in_channels, in_height, in_width]
  const auto &x_dims = x.dims();
  PADDLE_ENFORCE_EQ(
      x_dims.size() >= 2 && x_dims.size() <= 5,
      true,
      phi::errors::InvalidArgument(
          "The size of input's dimensions should be between 2 and 5"
          "But received: the size of input's dimensions is [%d]",
          x_dims.size()));

  ctx.template Alloc<T>(y);
  int N, C, H, W, D;
  phi::funcs::ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

  auto dtype = paddle::platform::CudnnDataType<T>::type;

#ifdef PADDLE_WITH_HIP
  auto compute_format =
      data_layout == DataLayout::kNHWC ? DataLayout::kNHWC : DataLayout::kNCHW;

// TODO(wangran16): wait for MIOpen to improve the performance of BN
// HIP do not support compute format of NHWC
// auto compute_format = DataLayout::kNCHW;
#else
  const bool fast_nhwc_batch_norm =
      test_mode ||
      (dtype == HIPDNN_DATA_HALF && FLAGS_cudnn_batchnorm_spatial_persistent);

  auto compute_format = fast_nhwc_batch_norm && data_layout == DataLayout::kNHWC
                            ? DataLayout::kNHWC
                            : DataLayout::kNCHW;
#endif

  DenseTensor transformed_x(x.type());
  DenseTensor transformed_y(y->type());

  if (data_layout == DataLayout::kNHWC && compute_format == DataLayout::kNCHW &&
      x_dims.size() > 2) {
    VLOG(3) << "Transform input tensor from NHWC to NCHW.";
    ResizeToChannelFirst<Context, T>(ctx, &x, &transformed_x);
    TransToChannelFirst<Context, T>(ctx, &x, &transformed_x);
    ResizeToChannelFirst<Context, T>(ctx, y, &transformed_y);
  } else {
    transformed_x.ShareDataWith(x);
    transformed_y.ShareDataWith(*y);
  }

// ------------------- cudnn descriptors ---------------------
#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// miopenTensorDescriptor_t data_desc_;
// miopenTensorDescriptor_t bn_param_desc_;
// miopenBatchNormMode_t mode_;

// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenCreateTensorDescriptor(&data_desc_));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenCreateTensorDescriptor(&bn_param_desc_));
#else
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t bn_param_desc_;
  hipdnnBatchNormMode_t mode_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
#endif

  if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
    LOG(ERROR) << "Provided epsilon is smaller than "
               << "HIPDNN_BN_MIN_EPSILON. Setting it to "
               << "HIPDNN_BN_MIN_EPSILON instead.";
  }
  epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// mode_ = miopenBNSpatial;
#elif CUDNN_VERSION_MIN(7, 0, 1)
  if (FLAGS_cudnn_batchnorm_spatial_persistent) {
    mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
  } else if (H == 1 && W == 1) {
    mode_ = HIPDNN_BATCHNORM_PER_ACTIVATION;
  } else {
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
  }
#else
  if (H == 1 && W == 1) {
    mode_ = HIPDNN_BATCHNORM_PER_ACTIVATION;
  } else {
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
  }
#endif  // CUDNN_VERSION_MIN(7, 0, 1)

  VLOG(3) << "Setting descriptors.";
  std::vector<int> dims;
  std::vector<int> strides;
  if (compute_format == DataLayout::kNCHW) {
    dims = {N, C, H, W, D};
    strides = {C * H * W * D, H * W * D, W * D, D, 1};
  } else {
    dims = {N, C, H, W, D};
    strides = {H * W * D * C, 1, W * D * C, D * C, C};
  }

#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::miopenSetTensorDescriptor(
//     data_desc_, CudnnDataType<T>::type,
//     x_dims.size() > 3 ? x_dims.size() : 4, const_cast<int *>(dims.data()),
//     const_cast<int *>(strides.data())));
// Note: PERSISTENT not implemented for inference
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDeriveBNTensorDescriptor(
//         bn_param_desc_, data_desc_, test_mode ? miopenBNSpatial : mode_));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_,
          CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4,
          dims.data(),
          strides.data()));
  // Note: PERSISTENT not implemented for inference
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDeriveBNTensorDescriptor(
          bn_param_desc_,
          data_desc_,
          test_mode ? HIPDNN_BATCHNORM_SPATIAL : mode_));
#endif

  auto handle = ctx.cudnn_handle();

  // Now, depending on whether we are running test or not, we have two paths.
  // It is training mode when it's not reference AND not using pre-trained
  // model.
  bool training = !test_mode && !use_global_stats;
  if (!training) {
    // only when test we use input to do computation.
    const auto *est_mean = &mean;
    const auto *est_var = &variance;
    // Run inference mode.
    PADDLE_ENFORCE_EQ(
        est_mean->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The size of mean's dimensions must equal to 1."
            "But received: the size of mean's dimensions mean is [%d],"
            "the dimensions of mean is [%s].",
            est_mean->dims().size(),
            est_mean->dims()));
    PADDLE_ENFORCE_EQ(
        est_var->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The size of variance's dimensions must equal to 1."
            "But received: the size of variance's dimensions is [%d],"
            "the dimensions of variance is [%s].",
            est_var->dims().size(),
            est_var->dims()));
    PADDLE_ENFORCE_EQ(
        est_mean->dims()[0],
        C,
        phi::errors::InvalidArgument(
            "The first dimension of mean must equal to the number of "
            "Channels, which is [%d]. But received: the first dimension"
            "of mean is [%d], the dimensions of mean is [%s].",
            C,
            est_mean->dims()[0],
            est_mean->dims()));
    PADDLE_ENFORCE_EQ(
        est_var->dims()[0],
        C,
        phi::errors::InvalidArgument(
            "The first dimension of variance must equal to the number"
            "of Channels, which is [%d]. But received: the first dimension of"
            "variance is [%d], the dimensions of variance is [%s].",
            C,
            est_var->dims()[0],
            est_var->dims()));

#ifdef PADDLE_WITH_HIP
    const int block_size = 256;
    const int grid_size = (N * C * H * W * D + block_size - 1) / block_size;
    if (compute_format == DataLayout::kNCHW) {
      BNForwardInference<T, DataLayout::kNCHW>
          <<<grid_size, block_size, 0, ctx.stream()>>>(
              transformed_x.template data<T>(),
              est_mean->template data<BatchNormParamType<T>>(),
              est_var->template data<BatchNormParamType<T>>(),
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              C,
              N,
              H * W * D,
              epsilon,
              transformed_y.template data<T>());
    } else {
      BNForwardInference<T, DataLayout::kNHWC>
          <<<grid_size, block_size, 0, ctx.stream()>>>(
              transformed_x.template data<T>(),
              est_mean->template data<BatchNormParamType<T>>(),
              est_var->template data<BatchNormParamType<T>>(),
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              C,
              N,
              H * W * D,
              epsilon,
              transformed_y.template data<T>());
    }
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenBatchNormalizationForwardInference(
//         handle, miopenBNSpatial,
//         const_cast<void *>(
//             static_cast<const void *>(CudnnDataType<T>::kOne())),
//         const_cast<void *>(
//             static_cast<const void *>(CudnnDataType<T>::kZero())),
//         data_desc_,
//         static_cast<const void *>(transformed_x.template data<T>()),
//         data_desc_,
//         static_cast<void *>(
//             transformed_y.template mutable_data<T>(ctx.GetPlace())),
//         bn_param_desc_,
//         const_cast<void *>(static_cast<const void *>(
//             scale->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             bias->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             est_mean->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             est_var->template data<BatchNormParamType<T>>())),
//         epsilon));
#else
    const bool use_native_kernel =
        (x_dims.size() == 2 ||
         (x_dims.size() == 3 && N >= CUDNN_SPATIAL_THRESHOLD_EVAL));
    if (use_native_kernel) {
      const int block_size = 256;
      const int grid_size = (N * C * H * W * D + block_size - 1) / block_size;
      if (compute_format == DataLayout::kNCHW) {
        BNForwardInference<T, DataLayout::kNCHW>
            <<<grid_size, block_size, 0, ctx.stream()>>>(
                transformed_x.template data<T>(),
                est_mean->template data<BatchNormParamType<T>>(),
                est_var->template data<BatchNormParamType<T>>(),
                scale.template data<BatchNormParamType<T>>(),
                bias.template data<BatchNormParamType<T>>(),
                C,
                N,
                H * W * D,
                epsilon,
                transformed_y.template data<T>());
      } else {
        if (x_dims.size() == 2) {
          DenseTensor inv_var = phi::Empty<BatchNormParamType<T>>(ctx, {C});
          auto *inv_var_ptr = inv_var.data<BatchNormParamType<T>>();
          const int threads = 512 > C ? C : 512;
          const int blocks = (C + 511) / 512;
          InverseVariance<T><<<blocks, threads>>>(
              est_var->template data<BatchNormParamType<T>>(),
              epsilon,
              C,
              inv_var_ptr);
          BN1DForwardInference<T, DataLayout::kNHWC>
              <<<grid_size, block_size, 0, ctx.stream()>>>(
                  transformed_x.template data<T>(),
                  est_mean->template data<BatchNormParamType<T>>(),
                  // est_var->template data<BatchNormParamType<T>>(),
                  inv_var_ptr,
                  scale.template data<BatchNormParamType<T>>(),
                  bias.template data<BatchNormParamType<T>>(),
                  C,
                  N,
                  H * W * D,
                  epsilon,
                  transformed_y.template data<T>());
        } else {
          BNForwardInference<T, DataLayout::kNHWC>
              <<<grid_size, block_size, 0, ctx.stream()>>>(
                  transformed_x.template data<T>(),
                  est_mean->template data<BatchNormParamType<T>>(),
                  est_var->template data<BatchNormParamType<T>>(),
                  scale.template data<BatchNormParamType<T>>(),
                  bias.template data<BatchNormParamType<T>>(),
                  C,
                  N,
                  H * W * D,
                  epsilon,
                  transformed_y.template data<T>());
        }
      }
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(
          paddle::platform::dynload::hipdnnBatchNormalizationForwardInference(
              handle,
              // Note: PERSISTENT not implemented for inference
              HIPDNN_BATCHNORM_SPATIAL,
              CudnnDataType<T>::kOne(),
              CudnnDataType<T>::kZero(),
              data_desc_,
              transformed_x.template data<T>(),
              data_desc_,
              ctx.template Alloc<T>(&transformed_y),
              bn_param_desc_,
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              est_mean->template data<BatchNormParamType<T>>(),
              est_var->template data<BatchNormParamType<T>>(),
              epsilon));
    }
#endif
  } else {
    // if MomentumTensor is set, use MomentumTensor value, momentum
    // is only used in this training branch

    // need to solve here
    // if (ctx.HasInput("MomentumTensor")) {
    //   const auto *mom_tensor = MomentumTensor;
    //   DenseTensor mom_cpu;
    //   paddle::framework::TensorCopySync(*mom_tensor, platform::CPUPlace(),
    //                                     &mom_cpu);
    //   momentum = mom_cpu.data<float>()[0];
    // }

    // Run training mode.
    // obtain running mean and running inv var, and there is no need
    // to initialize them.
    ctx.template Alloc<BatchNormParamType<T>>(mean_out);
    ctx.template Alloc<BatchNormParamType<T>>(variance_out);

    ctx.template Alloc<BatchNormParamType<T>>(saved_mean);
    ctx.template Alloc<BatchNormParamType<T>>(saved_variance);

    if ((N * H * W * D) == 1) {
      // Only 1 element in normalization dimension,
      // skip the batch norm calculation, let y = x.
      paddle::framework::TensorCopy(x, ctx.GetPlace(), y);
    } else {
      double this_factor = 1. - momentum;
#ifdef PADDLE_WITH_HIP
      const int num = transformed_x.numel();
      const int block = 256;
      const int max_threads = ctx.GetMaxPhysicalThreadCount();
      const int max_blocks = std::max(max_threads / block, 1);
      const int grid = std::min(C, max_blocks);
      if (compute_format == DataLayout::kNCHW) {
        BNForwardTraining<T, block, DataLayout::kNCHW>
            <<<grid, block, 0, ctx.stream()>>>(
                transformed_x.template data<T>(),
                scale.template data<BatchNormParamType<T>>(),
                bias.template data<BatchNormParamType<T>>(),
                C,
                N,
                H * W * D,
                epsilon,
                this_factor,
                transformed_y.template data<T>(),
                mean_out->template data<BatchNormParamType<T>>(),
                variance_out->template data<BatchNormParamType<T>>(),
                saved_mean->template data<BatchNormParamType<T>>(),
                saved_variance->template data<BatchNormParamType<T>>());
      } else {
        BNForwardTraining<T, block, DataLayout::kNHWC>
            <<<grid, block, 0, ctx.stream()>>>(
                transformed_x.template data<T>(),
                scale.template data<BatchNormParamType<T>>(),
                bias.template data<BatchNormParamType<T>>(),
                C,
                N,
                H * W * D,
                epsilon,
                this_factor,
                transformed_y.template data<T>(),
                mean_out->template data<BatchNormParamType<T>>(),
                variance_out->template data<BatchNormParamType<T>>(),
                saved_mean->template data<BatchNormParamType<T>>(),
                saved_variance->template data<BatchNormParamType<T>>());
      }
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenBatchNormalizationForwardTraining(
//         handle, mode_, const_cast<void *>(static_cast<const void *>(
//                            CudnnDataType<T>::kOne())),
//         const_cast<void *>(
//             static_cast<const void *>(CudnnDataType<T>::kZero())),
//         data_desc_,
//         static_cast<const void *>(transformed_x.template data<T>()),
//         data_desc_,
//         static_cast<void *>(
//             transformed_y.template mutable_data<T>(ctx.GetPlace())),
//         bn_param_desc_,
//         const_cast<void *>(static_cast<const void *>(
//             scale->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             bias->template data<BatchNormParamType<T>>())),
//         this_factor,
//         static_cast<void *>(
//             mean_out->template mutable_data<BatchNormParamType<T>>(
//                 ctx.GetPlace())),
//         static_cast<void *>(variance_out->template mutable_data<
//                             BatchNormParamType<T>>(ctx.GetPlace())),
//         epsilon,
//         static_cast<void *>(
//             saved_mean->template mutable_data<BatchNormParamType<T>>(
//                 ctx.GetPlace())),
//         static_cast<void *>(saved_variance->template mutable_data<
//                             BatchNormParamType<T>>(ctx.GetPlace()))));
#else
      // const size_t CUDNN_PER_ACTIVATION_THRESHOLD = 131070;
      const bool use_native_kernel =
          ((x_dims.size() == 2 && N >= CUDNN_PER_ACTIVATION_THRESHOLD) ||
           (x_dims.size() == 3 && N >= CUDNN_SPATIAL_THRESHOLD_TRAIN));
      if (use_native_kernel) {
        dim3 block;
        dim3 grid;
        const int block_size = 512;
        const int MAX_GRID_SIZE = 128;
        const int WARP_SIZE = 32;

        // init intermediate storage
        DenseTensor block_data_tensor;
        DenseTensor flag_tensor;
        DenseTensor compute_mean_tensor =
            phi::Empty<BatchNormParamType<T>, Context>(ctx, {C});
        DenseTensor compute_inv_var_tensor =
            phi::Empty<BatchNormParamType<T>, Context>(ctx, {C});

        BatchNormParamType<T> *block_data_ptr = nullptr;
        int *flag_ptr = nullptr;

        if (x_dims.size() != 2 && compute_format == DataLayout::kNCHW) {
          // init block&grid config
          int block_x =
              std::min(phi::funcs::details::GetLastPow2(H * W * D), block_size);
          int block_y = std::min(phi::funcs::details::GetLastPow2(C),
                                 block_size / block_x);

          if (block_x * block_y != block_size) {
            block_x =
                std::min(phi::funcs::details::GetLastPow2(N * H * W * D / 16),
                         block_size / block_y);
          }

          int grid_x =
              std::min((N * H * W * D + block_x * 16 - 1) / (block_x * 16),
                       MAX_GRID_SIZE);
          int grid_y = (C + block_y - 1) / block_y;

          block.x = block_x;
          block.y = block_y;
          grid.x = grid_x;
          grid.y = grid_y;

          if (grid.x > 1) {
            block_data_tensor = phi::Empty<BatchNormParamType<T>, Context>(
                ctx, {2 * C * grid.x});
            flag_tensor = phi::Empty<int, Context>(ctx, {grid.y});

            block_data_ptr = block_data_tensor.data<BatchNormParamType<T>>();
            flag_ptr = flag_tensor.data<int>();
            funcs::SetConstant<Context, int> set_zero;
            set_zero(ctx, &flag_tensor, static_cast<int>(0));
          }
          BNForwardTraining2DCompStat<T, block_size>
              <<<grid, block, 0, ctx.stream()>>>(
                  transformed_x.template data<T>(),
                  scale.template data<BatchNormParamType<T>>(),
                  bias.template data<BatchNormParamType<T>>(),
                  C,
                  N,
                  H * W * D,
                  epsilon,
                  this_factor,
                  transformed_y.template data<T>(),
                  mean_out->template data<BatchNormParamType<T>>(),
                  variance_out->template data<BatchNormParamType<T>>(),
                  saved_mean->template data<BatchNormParamType<T>>(),
                  saved_variance->template data<BatchNormParamType<T>>(),
                  compute_mean_tensor.data<BatchNormParamType<T>>(),
                  compute_inv_var_tensor.data<BatchNormParamType<T>>(),
                  block_data_ptr,
                  flag_ptr);

          BNForwardTraining2DWriteRes<T><<<grid, block, 0, ctx.stream()>>>(
              transformed_x.template data<T>(),
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              C,
              N,
              H * W * D,
              transformed_y.template data<T>(),
              compute_mean_tensor.data<BatchNormParamType<T>>(),
              compute_inv_var_tensor.data<BatchNormParamType<T>>());
        } else {
          // init block&grid config
          int block_x =
              std::min(phi::funcs::details::GetLastPow2(C), WARP_SIZE);
          int block_y =
              std::min(phi::funcs::details::GetLastPow2(N * H * W * D / 16),
                       block_size / block_x);
          if (block_x * block_y != block_size) {
            block_x = std::min(phi::funcs::details::GetLastPow2(C),
                               block_size / block_y);
          }
          int grid_x = (C + block_x - 1) / block_x;
          int grid_y =
              std::min((N * H * W * D + block_y * 16 - 1) / (block_y * 16),
                       MAX_GRID_SIZE);

          block.x = block_x;
          block.y = block_y;
          grid.x = grid_x;
          grid.y = grid_y;

          if (grid.y > 1) {
            block_data_tensor = phi::Empty<BatchNormParamType<T>, Context>(
                ctx, {2 * C * grid.y});
            flag_tensor = phi::Empty<int, Context>(ctx, {grid.x});

            block_data_ptr = block_data_tensor.data<BatchNormParamType<T>>();
            flag_ptr = flag_tensor.data<int>();
            funcs::SetConstant<Context, int> set_zero;
            set_zero(ctx, &flag_tensor, static_cast<int>(0));
          }
          BNForwardTraining2DChannelLastCompStat<T, block_size>
              <<<grid, block, 0, ctx.stream()>>>(
                  transformed_x.template data<T>(),
                  scale.template data<BatchNormParamType<T>>(),
                  bias.template data<BatchNormParamType<T>>(),
                  C,
                  N,
                  H * W * D,
                  epsilon,
                  this_factor,
                  transformed_y.template data<T>(),
                  mean_out->template data<BatchNormParamType<T>>(),
                  variance_out->template data<BatchNormParamType<T>>(),
                  saved_mean->template data<BatchNormParamType<T>>(),
                  saved_variance->template data<BatchNormParamType<T>>(),
                  compute_mean_tensor.data<BatchNormParamType<T>>(),
                  compute_inv_var_tensor.data<BatchNormParamType<T>>(),
                  block_data_ptr,
                  flag_ptr);

          BNForwardTraining2DChannelLastWriteRes<T>
              <<<grid, block, 0, ctx.stream()>>>(
                  transformed_x.template data<T>(),
                  scale.template data<BatchNormParamType<T>>(),
                  bias.template data<BatchNormParamType<T>>(),
                  C,
                  N,
                  H * W * D,
                  transformed_y.template data<T>(),
                  compute_mean_tensor.data<BatchNormParamType<T>>(),
                  compute_inv_var_tensor.data<BatchNormParamType<T>>());
        }
      } else {
#if CUDNN_VERSION_MIN(7, 4, 1)
        size_t workspace_size = 0;
        size_t reserve_space_size = 0;
        void *reserve_space_ptr = nullptr;
        void *workspace_ptr = nullptr;
        DenseTensor workspace_tensor;
        DenseTensor reserve_space_tensor;
        // Create reserve space and workspace for batch norm.
        // Create tensor for each batchnorm op, it will be used in the
        // backward. Thus this tensor shouldn't be temp.
        // auto *reserve_space = ctx.Output<Tensor>("ReserveSpace");
        if (reserve_space == nullptr) {
          reserve_space = &reserve_space_tensor;
        }
        PADDLE_ENFORCE_NOT_NULL(
            reserve_space,
            phi::errors::NotFound(
                "The argument ReserveSpace of batch_norm op is not found."));
        // --------------- cudnn batchnorm workspace ---------------
        PADDLE_ENFORCE_GPU_SUCCESS(
            paddle::platform::dynload::
                cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
                    /*handle=*/handle,
                    /*mode=*/mode_,
                    /*bnIps=*/CUDNN_BATCHNORM_OPS_BN,
                    /*xDesc=*/data_desc_,
                    /*zDesc=*/nullptr,
                    /*yDesc=*/data_desc_,
                    /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                    /*activationDesc=*/nullptr,
                    /*sizeInBytes=*/&workspace_size));

        // -------------- cudnn batchnorm reserve space --------------
        PADDLE_ENFORCE_GPU_SUCCESS(
            paddle::platform::dynload::
                cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
                    /*handle=*/handle,
                    /*mode=*/mode_,
                    /*bnOps=*/CUDNN_BATCHNORM_OPS_BN,
                    /*activationDesc=*/nullptr,
                    /*xDesc=*/data_desc_,
                    /*sizeInBytes=*/&reserve_space_size));

        reserve_space->Resize({static_cast<int64_t>(reserve_space_size)});
        reserve_space_ptr =
            static_cast<void *>(ctx.template Alloc<uint8_t>(reserve_space));
        workspace_tensor.Resize({static_cast<int64_t>(workspace_size)});
        workspace_ptr =
            static_cast<void *>(ctx.template Alloc<uint8_t>(&workspace_tensor));
        PADDLE_ENFORCE_GPU_SUCCESS(
            paddle::platform::dynload::cudnnBatchNormalizationForwardTrainingEx(
                handle,
                mode_,
                CUDNN_BATCHNORM_OPS_BN,
                CudnnDataType<T>::kOne(),
                CudnnDataType<T>::kZero(),
                data_desc_,
                transformed_x.template data<T>(),
                nullptr,
                nullptr,
                data_desc_,
                transformed_y.template data<T>(),
                bn_param_desc_,
                scale.template data<BatchNormParamType<T>>(),
                bias.template data<BatchNormParamType<T>>(),
                this_factor,
                ctx.template Alloc<BatchNormParamType<T>>(mean_out),
                ctx.template Alloc<BatchNormParamType<T>>(variance_out),
                epsilon,
                ctx.template Alloc<BatchNormParamType<T>>(saved_mean),
                ctx.template Alloc<BatchNormParamType<T>>(saved_variance),
                nullptr,
                workspace_ptr,
                workspace_size,
                reserve_space_ptr,
                reserve_space_size));
#else
        PADDLE_ENFORCE_GPU_SUCCESS(
            paddle::platform::dynload::hipdnnBatchNormalizationForwardTraining(
                handle,
                mode_,
                CudnnDataType<T>::kOne(),
                CudnnDataType<T>::kZero(),
                data_desc_,
                transformed_x.template data<T>(),
                data_desc_,
                ctx.template Alloc<T>(&transformed_y),
                bn_param_desc_,
                scale.template data<BatchNormParamType<T>>(),
                bias.template data<BatchNormParamType<T>>(),
                this_factor,
                ctx.template Alloc<BatchNormParamType<T>>(mean_out),
                ctx.template Alloc<BatchNormParamType<T>>(variance_out),
                epsilon,
                ctx.template Alloc<BatchNormParamType<T>>(saved_mean),
                ctx.template Alloc<BatchNormParamType<T>>(saved_variance)));
#endif  // CUDNN_VERSION_MIN(7, 4, 1)
      }
#endif
    }
  }

  if (data_layout == DataLayout::kNHWC && compute_format == DataLayout::kNCHW &&
      x_dims.size() > 2) {
    VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
    TransToChannelLast<Context, T>(ctx, &transformed_y, y);
  }
#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// clean when exit.
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDestroyTensorDescriptor(data_desc_));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDestroyTensorDescriptor(bn_param_desc_));
#else
  // clean when exit.
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
#endif
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(batch_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::BatchNormKernel,
                   float,
                   phi::dtype::float16) {
  kernel->InputAt(1).SetDataType(phi::DataType::FLOAT32);
  kernel->InputAt(2).SetDataType(phi::DataType::FLOAT32);
  kernel->InputAt(3).SetDataType(phi::DataType::FLOAT32);
  kernel->InputAt(4).SetDataType(phi::DataType::FLOAT32);
  kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
  kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
  kernel->OutputAt(3).SetDataType(phi::DataType::FLOAT32);
  kernel->OutputAt(4).SetDataType(phi::DataType::FLOAT32);
}
#else
PD_REGISTER_KERNEL(batch_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::BatchNormKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  if (kernel_key.dtype() == phi::DataType::FLOAT16) {
    kernel->InputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->InputAt(2).SetDataType(phi::DataType::FLOAT32);
    kernel->InputAt(3).SetDataType(phi::DataType::FLOAT32);
    kernel->InputAt(4).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(3).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(4).SetDataType(phi::DataType::FLOAT32);
  }
}

#endif
