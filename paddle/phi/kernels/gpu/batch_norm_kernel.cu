#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/batch_norm_kernel.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"

#include "paddle/fluid/operators/norm_utils.cu.h"
#include "paddle/fluid/operators/norm_utils.h"

#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/operators/layout_utils.h"
#include "paddle/fluid/platform/enforce.h"

#include "paddle/fluid/platform/flags.h"
#include "paddle/phi/kernels/gpu/batch_norm_utils.h"

#ifdef __HIPCC__
#define LAUNCH_BOUNDS(BlockDim) __launch_bounds__(BlockDim)
#else
#define LAUNCH_BOUNDS(BlockDim)
#endif

DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace phi {

template <typename T>
using CudnnDataType = paddle::platform::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

template <typename T, phi::DataLayout layout>
static __global__ void BNForwardInference(const T *x,
                                          const BatchNormParamType<T> *mean,
                                          const BatchNormParamType<T> *variance,
                                          const BatchNormParamType<T> *scale,
                                          const BatchNormParamType<T> *bias,
                                          const int C,
                                          const int N,
                                          const int HxW,
                                          const double epsilon,
                                          T *y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int num = N * C * HxW;
  for (int i = gid; i < num; i += stride) {
    const int c = layout == phi::DataLayout::kNCHW ? i / HxW % C : i % C;
    BatchNormParamType<T> x_sub_mean =
        static_cast<BatchNormParamType<T>>(x[i]) - mean[c];
    BatchNormParamType<T> inv_var = 1 / sqrt(variance[c] + epsilon);
    y[i] = static_cast<T>(scale[c] * x_sub_mean * inv_var + bias[c]);
  }
}

template <typename T, int BlockDim, phi::DataLayout layout>
static __global__ LAUNCH_BOUNDS(BlockDim) void BNForwardTraining(
    const T *x,
    const BatchNormParamType<T> *scale,
    const BatchNormParamType<T> *bias,
    const int C,
    const int N,
    const int HxW,
    const double epsilon,
    double exponentialAverageFactor,
    T *y,
    BatchNormParamType<T> *mean,
    BatchNormParamType<T> *variance,
    BatchNormParamType<T> *save_mean,
    BatchNormParamType<T> *save_inv_variance) {
  int outer_size = C;
  int inner_size = N * HxW;
  typedef hipcub::BlockReduce<BatchNormParamType<T>, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage mean_storage;
  __shared__ typename BlockReduce::TempStorage variance_storeage;
  __shared__ BatchNormParamType<T> mean_val;
  __shared__ BatchNormParamType<T> variance_val;
  __shared__ BatchNormParamType<T> inv_var_val;

  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    BatchNormParamType<T> x_sum = static_cast<BatchNormParamType<T>>(0);
    BatchNormParamType<T> x_square_sum = static_cast<BatchNormParamType<T>>(0);

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == phi::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      BatchNormParamType<T> x_i = static_cast<BatchNormParamType<T>>(x[index]);
      x_sum += x_i;
      x_square_sum += x_i * x_i;
    }
    x_sum = BlockReduce(mean_storage).Reduce(x_sum, hipcub::Sum());
    x_square_sum =
        BlockReduce(variance_storeage).Reduce(x_square_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      mean_val = x_sum / inner_size;
      variance_val = x_square_sum / inner_size - mean_val * mean_val;
      inv_var_val = 1 / sqrt(variance_val + epsilon);

      if (save_mean && save_inv_variance) {
        save_mean[i] = mean_val;
        save_inv_variance[i] = inv_var_val;
      }
      mean[i] = (1 - exponentialAverageFactor) * mean_val +
                exponentialAverageFactor * mean[i];
      variance[i] = (1 - exponentialAverageFactor) * variance_val +
                    exponentialAverageFactor * variance[i];
    }
    __syncthreads();

    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = layout == phi::DataLayout::kNCHW
                            ? (j / HxW * C + i) * HxW + j % HxW
                            : j * outer_size + i;
      BatchNormParamType<T> x_sub_mean =
          static_cast<BatchNormParamType<T>>(x[index]) - mean_val;
      y[index] = scale[i] * x_sub_mean * inv_var_val + bias[i];
    }
  }
}

template <typename T, typename Context>
void BatchNormKernel(const Context &ctx,
                     const DenseTensor &x,
                     const DenseTensor &scale,
                     const DenseTensor &bias,
                     const DenseTensor &mean,
                     const DenseTensor &variance,
                     float momentum,
                     float epsilon_f,
                     const std::string &data_layout_str,
                     bool is_test,
                     bool use_global_stats,
                     bool trainable_statistics,
                     bool fuse_with_relu,
                     DenseTensor *y,
                     DenseTensor *mean_out,
                     DenseTensor *variance_out,
                     DenseTensor *saved_mean,
                     DenseTensor *saved_variance,
                     DenseTensor *reserve_space) {
  double epsilon = epsilon_f;
  const bool trainable_stats = trainable_statistics;
  const DataLayout data_layout =
      paddle::framework::StringToDataLayout(data_layout_str);
  bool test_mode = is_test && (!trainable_stats);

  // Get the size for each dimension.
  // NCHW [batch_size, in_channels, in_height, in_width]
  const auto &x_dims = x.dims();
  PADDLE_ENFORCE_EQ(
      x_dims.size() >= 2 && x_dims.size() <= 5,
      true,
      phi::errors::InvalidArgument(
          "The size of input's dimensions should be between 2 and 5"
          "But received: the size of input's dimensions is [%d]",
          x_dims.size()));

  ctx.template Alloc<T>(y);
  int N, C, H, W, D;
  paddle::operators::ExtractNCWHD(x_dims, data_layout, &N, &C, &H, &W, &D);

  auto dtype = paddle::platform::CudnnDataType<T>::type;

#ifdef PADDLE_WITH_HIP
  auto compute_format =
      data_layout == DataLayout::kNHWC ? DataLayout::kNHWC : DataLayout::kNCHW;

// TODO(wangran16): wait for MIOpen to improve the performance of BN
// HIP do not support compute format of NHWC
// auto compute_format = DataLayout::kNCHW;
#else
  const bool fast_nhwc_batch_norm =
      test_mode ||
      (dtype == HIPDNN_DATA_HALF && FLAGS_cudnn_batchnorm_spatial_persistent);

  auto compute_format = fast_nhwc_batch_norm && data_layout == DataLayout::kNHWC
                            ? DataLayout::kNHWC
                            : DataLayout::kNCHW;
#endif

  DenseTensor transformed_x(x.type());
  DenseTensor transformed_y(y->type());

  if (data_layout == DataLayout::kNHWC && compute_format == DataLayout::kNCHW &&
      x_dims.size() > 2) {
    VLOG(3) << "Transform input tensor from NHWC to NCHW.";
    ResizeToChannelFirst<Context, T>(ctx, &x, &transformed_x);
    TransToChannelFirst<Context, T>(ctx, &x, &transformed_x);
    ResizeToChannelFirst<Context, T>(ctx, y, &transformed_y);
  } else {
    transformed_x.ShareDataWith(x);
    transformed_y.ShareDataWith(*y);
  }

// ------------------- cudnn descriptors ---------------------
#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// miopenTensorDescriptor_t data_desc_;
// miopenTensorDescriptor_t bn_param_desc_;
// miopenBatchNormMode_t mode_;

// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenCreateTensorDescriptor(&data_desc_));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenCreateTensorDescriptor(&bn_param_desc_));
#else
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t bn_param_desc_;
  hipdnnBatchNormMode_t mode_;

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnCreateTensorDescriptor(&bn_param_desc_));
#endif

  if (epsilon <= HIPDNN_BN_MIN_EPSILON - FLT_EPSILON) {
    LOG(ERROR) << "Provided epsilon is smaller than "
               << "HIPDNN_BN_MIN_EPSILON. Setting it to "
               << "HIPDNN_BN_MIN_EPSILON instead.";
  }
  epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);

#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// mode_ = miopenBNSpatial;
#elif CUDNN_VERSION_MIN(7, 0, 1)
  if (FLAGS_cudnn_batchnorm_spatial_persistent) {
    mode_ = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
  } else if (H == 1 && W == 1) {
    mode_ = HIPDNN_BATCHNORM_PER_ACTIVATION;
  } else {
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
  }
#else
  if (H == 1 && W == 1) {
    mode_ = HIPDNN_BATCHNORM_PER_ACTIVATION;
  } else {
    mode_ = HIPDNN_BATCHNORM_SPATIAL;
  }
#endif  // CUDNN_VERSION_MIN(7, 0, 1)

  VLOG(3) << "Setting descriptors.";
  std::vector<int> dims;
  std::vector<int> strides;
  if (compute_format == DataLayout::kNCHW) {
    dims = {N, C, H, W, D};
    strides = {C * H * W * D, H * W * D, W * D, D, 1};
  } else {
    dims = {N, C, H, W, D};
    strides = {H * W * D * C, 1, W * D * C, D * C, C};
  }

#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(platform::dynload::miopenSetTensorDescriptor(
//     data_desc_, CudnnDataType<T>::type,
//     x_dims.size() > 3 ? x_dims.size() : 4, const_cast<int *>(dims.data()),
//     const_cast<int *>(strides.data())));
// Note: PERSISTENT not implemented for inference
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDeriveBNTensorDescriptor(
//         bn_param_desc_, data_desc_, test_mode ? miopenBNSpatial : mode_));
#else
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnSetTensorNdDescriptor(
          data_desc_,
          CudnnDataType<T>::type,
          x_dims.size() > 3 ? x_dims.size() : 4,
          dims.data(),
          strides.data()));
  // Note: PERSISTENT not implemented for inference
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDeriveBNTensorDescriptor(
          bn_param_desc_,
          data_desc_,
          test_mode ? HIPDNN_BATCHNORM_SPATIAL : mode_));
#endif

  auto handle = ctx.cudnn_handle();

  // Now, depending on whether we are running test or not, we have two paths.
  // It is training mode when it's not reference AND not using pre-trained
  // model.
  bool training = !test_mode && !use_global_stats;
  if (!training) {
    // only when test we use input to do computation.
    const auto *est_mean = &mean;
    const auto *est_var = &variance;
    // Run inference mode.
    PADDLE_ENFORCE_EQ(
        est_mean->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The size of mean's dimensions must equal to 1."
            "But received: the size of mean's dimensions mean is [%d],"
            "the dimensions of mean is [%s].",
            est_mean->dims().size(),
            est_mean->dims()));
    PADDLE_ENFORCE_EQ(
        est_var->dims().size(),
        1UL,
        phi::errors::InvalidArgument(
            "The size of variance's dimensions must equal to 1."
            "But received: the size of variance's dimensions is [%d],"
            "the dimensions of variance is [%s].",
            est_var->dims().size(),
            est_var->dims()));
    PADDLE_ENFORCE_EQ(
        est_mean->dims()[0],
        C,
        phi::errors::InvalidArgument(
            "The first dimension of mean must equal to the number of "
            "Channels, which is [%d]. But received: the first dimension"
            "of mean is [%d], the dimensions of mean is [%s].",
            C,
            est_mean->dims()[0],
            est_mean->dims()));
    PADDLE_ENFORCE_EQ(
        est_var->dims()[0],
        C,
        phi::errors::InvalidArgument(
            "The first dimension of variance must equal to the number"
            "of Channels, which is [%d]. But received: the first dimension of"
            "variance is [%d], the dimensions of variance is [%s].",
            C,
            est_var->dims()[0],
            est_var->dims()));

#ifdef PADDLE_WITH_HIP
    const int block_size = 256;
    const int grid_size = (N * C * H * W * D + block_size - 1) / block_size;
    if (compute_format == DataLayout::kNCHW) {
      BNForwardInference<
          T,
          DataLayout::kNCHW><<<grid_size, block_size, 0, ctx.stream()>>>(
          transformed_x.template data<T>(),
          est_mean->template data<BatchNormParamType<T>>(),
          est_var->template data<BatchNormParamType<T>>(),
          scale.template data<BatchNormParamType<T>>(),
          bias.template data<BatchNormParamType<T>>(),
          C,
          N,
          H * W * D,
          epsilon,
          transformed_y.template data<T>());
    } else {
      BNForwardInference<
          T,
          DataLayout::kNHWC><<<grid_size, block_size, 0, ctx.stream()>>>(
          transformed_x.template data<T>(),
          est_mean->template data<BatchNormParamType<T>>(),
          est_var->template data<BatchNormParamType<T>>(),
          scale.template data<BatchNormParamType<T>>(),
          bias.template data<BatchNormParamType<T>>(),
          C,
          N,
          H * W * D,
          epsilon,
          transformed_y.template data<T>());
    }
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenBatchNormalizationForwardInference(
//         handle, miopenBNSpatial,
//         const_cast<void *>(
//             static_cast<const void *>(CudnnDataType<T>::kOne())),
//         const_cast<void *>(
//             static_cast<const void *>(CudnnDataType<T>::kZero())),
//         data_desc_,
//         static_cast<const void *>(transformed_x.template data<T>()),
//         data_desc_,
//         static_cast<void *>(
//             transformed_y.template mutable_data<T>(ctx.GetPlace())),
//         bn_param_desc_,
//         const_cast<void *>(static_cast<const void *>(
//             scale->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             bias->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             est_mean->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             est_var->template data<BatchNormParamType<T>>())),
//         epsilon));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(
        paddle::platform::dynload::hipdnnBatchNormalizationForwardInference(
            handle,
            // Note: PERSISTENT not implemented for inference
            HIPDNN_BATCHNORM_SPATIAL,
            CudnnDataType<T>::kOne(),
            CudnnDataType<T>::kZero(),
            data_desc_,
            transformed_x.template data<T>(),
            data_desc_,
            ctx.template Alloc<T>(&transformed_y),
            bn_param_desc_,
            scale.template data<BatchNormParamType<T>>(),
            bias.template data<BatchNormParamType<T>>(),
            est_mean->template data<BatchNormParamType<T>>(),
            est_var->template data<BatchNormParamType<T>>(),
            epsilon));
#endif
  } else {
    // if MomentumTensor is set, use MomentumTensor value, momentum
    // is only used in this training branch

    // need to solve here
    // if (ctx.HasInput("MomentumTensor")) {
    //   const auto *mom_tensor = MomentumTensor;
    //   DenseTensor mom_cpu;
    //   paddle::framework::TensorCopySync(*mom_tensor, platform::CPUPlace(),
    //                                     &mom_cpu);
    //   momentum = mom_cpu.data<float>()[0];
    // }

    // Run training mode.
    // obtain running mean and running inv var, and there is no need
    // to initialize them.
    ctx.template Alloc<BatchNormParamType<T>>(mean_out);
    ctx.template Alloc<BatchNormParamType<T>>(variance_out);

    ctx.template Alloc<BatchNormParamType<T>>(saved_mean);
    ctx.template Alloc<BatchNormParamType<T>>(saved_variance);

    if ((N * H * W * D) == 1) {
      // Only 1 element in normalization dimension,
      // skip the batch norm calculation, let y = x.
      paddle::framework::TensorCopy(x, ctx.GetPlace(), y);
    } else {
      double this_factor = 1. - momentum;

      bool called = false;
#if CUDNN_VERSION_MIN(7, 4, 1)
      called = true;
      size_t workspace_size = 0;
      size_t reserve_space_size = 0;
      void *reserve_space_ptr = nullptr;
      void *workspace_ptr = nullptr;
      DenseTensor workspace_tensor;
      DenseTensor reserve_space_tensor;
      // Create reserve space and workspace for batch norm.
      // Create tensor for each batchnorm op, it will be used in the
      // backward. Thus this tensor shouldn't be temp.
      // auto *reserve_space = ctx.Output<Tensor>("ReserveSpace");
      if (reserve_space == nullptr) {
        reserve_space = &reserve_space_tensor;
      }
      PADDLE_ENFORCE_NOT_NULL(
          reserve_space,
          phi::errors::NotFound(
              "The argument ReserveSpace of batch_norm op is not found."));
      // --------------- cudnn batchnorm workspace ---------------
      PADDLE_ENFORCE_GPU_SUCCESS(
          paddle::platform::dynload::
              cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
                  /*handle=*/handle,
                  /*mode=*/mode_,
                  /*bnIps=*/CUDNN_BATCHNORM_OPS_BN,
                  /*xDesc=*/data_desc_,
                  /*zDesc=*/nullptr,
                  /*yDesc=*/data_desc_,
                  /*bnScaleBiasMeanVarDesc=*/bn_param_desc_,
                  /*activationDesc=*/nullptr,
                  /*sizeInBytes=*/&workspace_size));

      // -------------- cudnn batchnorm reserve space --------------
      PADDLE_ENFORCE_GPU_SUCCESS(
          paddle::platform::dynload::
              cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
                  /*handle=*/handle,
                  /*mode=*/mode_,
                  /*bnOps=*/CUDNN_BATCHNORM_OPS_BN,
                  /*activationDesc=*/nullptr,
                  /*xDesc=*/data_desc_,
                  /*sizeInBytes=*/&reserve_space_size));

      reserve_space->Resize({static_cast<int64_t>(reserve_space_size)});
      reserve_space_ptr = ctx.template Alloc<uint8_t>(reserve_space);
      workspace_tensor.Resize({static_cast<int64_t>(workspace_size)});
      workspace_ptr = ctx.template Alloc<uint8_t>(&workspace_tensor);
      PADDLE_ENFORCE_GPU_SUCCESS(
          paddle::platform::dynload::cudnnBatchNormalizationForwardTrainingEx(
              handle,
              mode_,
              CUDNN_BATCHNORM_OPS_BN,
              CudnnDataType<T>::kOne(),
              CudnnDataType<T>::kZero(),
              data_desc_,
              transformed_x.template data<T>(),
              nullptr,
              nullptr,
              data_desc_,
              transformed_y.template data<T>(),
              bn_param_desc_,
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              this_factor,
              ctx.template Alloc<BatchNormParamType<T>>(mean_out),
              ctx.template Alloc<BatchNormParamType<T>>(variance_out),
              epsilon,
              ctx.template Alloc<BatchNormParamType<T>>(saved_mean),
              ctx.template Alloc<BatchNormParamType<T>>(saved_variance),
              nullptr,
              workspace_ptr,
              workspace_size,
              reserve_space_ptr,
              reserve_space_size));
#endif  // CUDNN_VERSION_MIN(7, 4, 1)
      if (!called) {
#ifdef PADDLE_WITH_HIP
        const int num = transformed_x.numel();
        const int block = 256;
        const int max_threads = ctx.GetMaxPhysicalThreadCount();
        const int max_blocks = std::max(max_threads / block, 1);
        const int grid = std::min(C, max_blocks);
        if (compute_format == DataLayout::kNCHW) {
          BNForwardTraining<
              T,
              block,
              DataLayout::kNCHW><<<grid, block, 0, ctx.stream()>>>(
              transformed_x.template data<T>(),
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              C,
              N,
              H * W * D,
              epsilon,
              this_factor,
              transformed_y.template data<T>(),
              mean_out->template data<BatchNormParamType<T>>(),
              variance_out->template data<BatchNormParamType<T>>(),
              saved_mean->template data<BatchNormParamType<T>>(),
              saved_variance->template data<BatchNormParamType<T>>());
        } else {
          BNForwardTraining<
              T,
              block,
              DataLayout::kNHWC><<<grid, block, 0, ctx.stream()>>>(
              transformed_x.template data<T>(),
              scale.template data<BatchNormParamType<T>>(),
              bias.template data<BatchNormParamType<T>>(),
              C,
              N,
              H * W * D,
              epsilon,
              this_factor,
              transformed_y.template data<T>(),
              mean_out->template data<BatchNormParamType<T>>(),
              variance_out->template data<BatchNormParamType<T>>(),
              saved_mean->template data<BatchNormParamType<T>>(),
              saved_variance->template data<BatchNormParamType<T>>());
        }
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenBatchNormalizationForwardTraining(
//         handle, mode_, const_cast<void *>(static_cast<const void *>(
//                            CudnnDataType<T>::kOne())),
//         const_cast<void *>(
//             static_cast<const void *>(CudnnDataType<T>::kZero())),
//         data_desc_,
//         static_cast<const void *>(transformed_x.template data<T>()),
//         data_desc_,
//         static_cast<void *>(
//             transformed_y.template mutable_data<T>(ctx.GetPlace())),
//         bn_param_desc_,
//         const_cast<void *>(static_cast<const void *>(
//             scale->template data<BatchNormParamType<T>>())),
//         const_cast<void *>(static_cast<const void *>(
//             bias->template data<BatchNormParamType<T>>())),
//         this_factor,
//         static_cast<void *>(
//             mean_out->template mutable_data<BatchNormParamType<T>>(
//                 ctx.GetPlace())),
//         static_cast<void *>(variance_out->template mutable_data<
//                             BatchNormParamType<T>>(ctx.GetPlace())),
//         epsilon,
//         static_cast<void *>(
//             saved_mean->template mutable_data<BatchNormParamType<T>>(
//                 ctx.GetPlace())),
//         static_cast<void *>(saved_variance->template mutable_data<
//                             BatchNormParamType<T>>(ctx.GetPlace()))));
#else
        PADDLE_ENFORCE_GPU_SUCCESS(
            paddle::platform::dynload::hipdnnBatchNormalizationForwardTraining(
                handle,
                mode_,
                CudnnDataType<T>::kOne(),
                CudnnDataType<T>::kZero(),
                data_desc_,
                transformed_x.template data<T>(),
                data_desc_,
                ctx.template Alloc<T>(&transformed_y),
                bn_param_desc_,
                scale.template data<BatchNormParamType<T>>(),
                bias.template data<BatchNormParamType<T>>(),
                this_factor,
                ctx.template Alloc<BatchNormParamType<T>>(mean_out),
                ctx.template Alloc<BatchNormParamType<T>>(variance_out),
                epsilon,
                ctx.template Alloc<BatchNormParamType<T>>(saved_mean),
                ctx.template Alloc<BatchNormParamType<T>>(saved_variance)));
#endif
      }
    }
  }

  if (data_layout == DataLayout::kNHWC && compute_format == DataLayout::kNCHW &&
      x_dims.size() > 2) {
    VLOG(3) << "Transform batchnorm output from NCHW to NHWC";
    TransToChannelLast<Context, T>(ctx, &transformed_y, y);
  }
#ifdef PADDLE_WITH_HIP
// TODO(wangran16): wait for MIOpen to improve the performance of BN
// clean when exit.
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDestroyTensorDescriptor(data_desc_));
// PADDLE_ENFORCE_GPU_SUCCESS(
//     platform::dynload::miopenDestroyTensorDescriptor(bn_param_desc_));
#else
  // clean when exit.
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(data_desc_));
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnDestroyTensorDescriptor(bn_param_desc_));
#endif
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(batch_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::BatchNormKernel,
                   float,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(batch_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::BatchNormKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  if (kernel_key.dtype() == phi::DataType::FLOAT16) {
    kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(3).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(4).SetDataType(phi::DataType::FLOAT32);
  }
}

#endif
