#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"

#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/reshape_kernel.h"

#ifdef PADDLE_WITH_FLASHATTN
#include "paddle/phi/backends/dynload/flashattn.h"
#endif

namespace phi {

template <typename T, typename Context>
void FlashAttnKernel(const Context& ctx,
                     const DenseTensor& q,
                     const DenseTensor& k,
                     const DenseTensor& v,
                     float dropout,
                     bool causal,
                     bool return_softmax,
                     DenseTensor* out,
                     DenseTensor* softmax_lse,
                     DenseTensor* softmax,
                     DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(out);

  hipStream_t stream = ctx.stream();
  bool is_bf16 = q.dtype() == DataType::BFLOAT16 ? true : false;

  // q,k,v [batch_size, seq_len, num_heads, head_dim]

  auto dims = q.dims();
  int64_t batch_size = dims[0];
  int64_t seq_len_q = dims[1];
  int64_t num_heads = dims[2];
  int64_t head_size = dims[3];

  int64_t seq_len_k = k.dims()[1];

  int64_t total_q = batch_size * seq_len_q;
  int64_t total_k = batch_size * seq_len_k;

  DenseTensor q_t_s =
      Reshape<T, Context>(ctx, q, {total_q, num_heads, head_size});
  DenseTensor k_t_s =
      Reshape<T, Context>(ctx, k, {total_k, num_heads, head_size});
  DenseTensor v_t_s =
      Reshape<T, Context>(ctx, v, {total_k, num_heads, head_size});

  // q,k,v [total_*, num_heads, head_dim]

  DenseTensor cu_seqlens_q;
  DenseTensor cu_seqlens_k;
  ArangeNullaryKernel<int32_t, Context>(
      ctx, 0, (batch_size + 1) * seq_len_q, seq_len_q, &cu_seqlens_q);
  ArangeNullaryKernel<int32_t, Context>(
      ctx, 0, (batch_size + 1) * seq_len_k, seq_len_k, &cu_seqlens_k);

  float scale = 1.0f / std::sqrt(head_size);
  int num_splits = 0;  // 0 for an internal heuristic, which is optimal
  bool zero_tensors = false;

  auto gen = ctx.GetGenerator();
  uint64_t inc = batch_size * num_heads * 32;
  auto seed_offset_pair = gen->IncrementOffset(inc);
  uint64_t seed = seed_offset_pair.first;
  uint64_t offset = seed_offset_pair.second;

  std::vector<int64_t> seed_offset_vec{int64_t(seed), int64_t(offset)};
  phi::TensorFromVector<int64_t>(seed_offset_vec, ctx, seed_offset);

  softmax_lse->Resize({batch_size, num_heads, seq_len_q});
  ctx.template Alloc<float>(softmax_lse);

  if (return_softmax) {
    // may allocate more space than *seq_len_k*
    int64_t blocksize_c = head_size > 64 ? 128 : 256;
    int64_t max_len_k_ =
        ((seq_len_k + blocksize_c - 1) / blocksize_c) * blocksize_c;
    int64_t max_len_k =
        seq_len_k <= 128 ? 128 : (seq_len_k <= 256 ? 256 : max_len_k_);
    softmax->Resize({batch_size, num_heads, seq_len_q, max_len_k});
    ctx.template Alloc<T>(softmax);
  }

  uint64_t workspace_size;

  // calculate workspace size before execution
  bool succ =
      phi::dynload::flash_attn_fwd(q_t_s.data(),
                                   k_t_s.data(),
                                   v_t_s.data(),
                                   nullptr,  // for calculation workspace size
                                   cu_seqlens_q.data(),
                                   cu_seqlens_k.data(),
                                   total_q,
                                   total_k,
                                   batch_size,
                                   num_heads,
                                   head_size,
                                   seq_len_q,
                                   seq_len_k,
                                   dropout,
                                   scale,
                                   zero_tensors,
                                   causal,
                                   is_bf16,
                                   num_splits,
                                   softmax_lse->data(),
                                   return_softmax ? softmax->data() : nullptr,
                                   nullptr,
                                   &workspace_size,
                                   stream,
                                   seed,
                                   offset);

  if (!succ) {
    PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
  }

  DenseTensor workspace;
  if (workspace_size > 0) {
    workspace = Empty<float>(ctx, {int64_t(workspace_size / sizeof(float))});
  }

  succ = phi::dynload::flash_attn_fwd(
      q_t_s.data(),
      k_t_s.data(),
      v_t_s.data(),
      out->data(),
      cu_seqlens_q.data(),
      cu_seqlens_k.data(),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      seq_len_q,
      seq_len_k,
      dropout,
      scale,
      zero_tensors,
      causal,
      is_bf16,
      num_splits,
      softmax_lse->data(),
      return_softmax ? softmax->data() : nullptr,
      workspace_size > 0 ? workspace.data() : nullptr,
      &workspace_size,
      stream,
      seed,
      offset);

  if (!succ) {
    PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
  }

#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
