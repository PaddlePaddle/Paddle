#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_kernel.h"

#include "glog/logging.h"  // For VLOG()
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/reshape_kernel.h"

#ifdef PADDLE_WITH_FLASHATTN
#include "paddle/phi/backends/dynload/flashattn.h"
#endif

DECLARE_bool(cudnn_deterministic);

namespace phi {

template <typename T, typename Context>
void FlashAttnUnpaddedKernel(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& cu_seqlens_q,
    const DenseTensor& cu_seqlens_k,
    const paddle::optional<DenseTensor>& fixed_seed_offset,
    int64_t max_seqlen_q,
    int64_t max_seqlen_k,
    float scale,
    float dropout,
    bool causal,
    bool return_softmax,
    bool is_test,
    const std::string& rng_name,
    DenseTensor* out,
    DenseTensor* softmax,
    DenseTensor* softmax_lse,
    DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  if (is_test) dropout = 0.0f;

  ctx.template Alloc<T>(out);

  hipStream_t stream = ctx.stream();
  bool is_bf16 = q.dtype() == DataType::BFLOAT16 ? true : false;

  // q,k,v [total_*, num_heads, head_dim]

  auto dims = q.dims();
  PADDLE_ENFORCE_EQ(
      dims.size(),
      3,
      phi::errors::InvalidArgument("flash_attn_raw receive input with dim "
                                   "[total_seq_len, num_heads, head_dim]"));

  const int total_q = dims[0];
  const int num_heads = dims[1];
  const int head_size = dims[2];

  const int total_k = k.dims()[0];
  const int num_heads_k = k.dims()[1];
  const int batch_size = cu_seqlens_q.numel() - 1;

  // TODO(umiswing): add shape check

  // fa-2 always sets zero_tensors to false
  const bool zero_tensors = false;

  uint64_t seed;
  uint64_t offset;

  if (fixed_seed_offset.get_ptr()) {
    const int64_t* fixed_seed_offset_data =
        fixed_seed_offset.get_ptr()->data<int64_t>();
    seed = static_cast<uint64_t>(fixed_seed_offset_data[0]);
    offset = static_cast<uint64_t>(fixed_seed_offset_data[1]);
  } else {
    uint64_t inc = batch_size * num_heads * 32;
    std::pair<uint64_t, uint64_t> seed_offset_pair;
    if (rng_name != "") {
      auto gen = phi::GetRandomSeedGenerator(rng_name);
      seed_offset_pair = gen->IncrementOffset(inc);
    } else {
      auto* gen = ctx.GetGenerator();
      seed_offset_pair = gen->IncrementOffset(inc);
    }
    seed = seed_offset_pair.first;
    offset = seed_offset_pair.second;
  }

  VLOG(4) << "FlashAttn fwd seed: " << seed << ", offset: " << offset;

  seed_offset->Resize({2});
  int64_t* seed_offset_data = ctx.template HostAlloc<int64_t>(seed_offset);
  seed_offset_data[0] = static_cast<int64_t>(seed);
  seed_offset_data[1] = static_cast<int64_t>(offset);

  auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
  const int head_size_rounded = round_multiple(head_size, 32);
  const int seqlen_q_rounded = round_multiple(max_seqlen_q, 128);
  const int seqlen_k_rounded = round_multiple(max_seqlen_k, 128);

  softmax_lse->Resize({batch_size, num_heads, max_seqlen_q});
  ctx.template Alloc<float>(softmax_lse);

  if (return_softmax) {
    // may allocate more space than *max_seqlen_k*
    softmax->Resize(
        {batch_size, num_heads, seqlen_q_rounded, seqlen_k_rounded});
    ctx.template Alloc<T>(softmax);
  }

  const bool succ = phi::dynload::flash_attn_varlen_fwd(
      q.data(),
      k.data(),
      v.data(),
      out->data(),
      cu_seqlens_q.data<int32_t>(),
      cu_seqlens_k.data<int32_t>(),
      return_softmax ? softmax->data() : nullptr,
      softmax_lse->data(),
      batch_size,
      max_seqlen_q,
      max_seqlen_k,
      seqlen_q_rounded,
      seqlen_k_rounded,
      num_heads,
      num_heads_k,
      head_size,
      head_size_rounded,
      dropout,
      scale,
      causal,
      return_softmax,
      is_bf16,
      stream,
      seed,
      offset);

  if (!succ) {
    PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
  }

#endif
}

template <typename T, typename Context>
void FlashAttnKernel(const Context& ctx,
                     const DenseTensor& q,
                     const DenseTensor& k,
                     const DenseTensor& v,
                     const paddle::optional<DenseTensor>& fixed_seed_offset,
                     float dropout,
                     bool causal,
                     bool return_softmax,
                     bool is_test,
                     const std::string& rng_name,
                     DenseTensor* out,
                     DenseTensor* softmax,
                     DenseTensor* softmax_lse,
                     DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [batch_size, seq_len, num_heads, head_dim]

  auto dims = q.dims();
  PADDLE_ENFORCE_EQ(dims.size(),
                    4,
                    phi::errors::InvalidArgument(
                        "flash_attn receive input with dim "
                        "[batch_size, seq_len, num_heads, head_dim]"));

  const int batch_size = dims[0];
  const int seqlen_q = dims[1];
  const int num_heads = dims[2];
  const int head_size = dims[3];
  const int seqlen_k = k.dims()[1];
  const int num_heads_k = k.dims()[2];

  // TODO(umiswing): Add check shape

  const float scale = 1.0f / std::sqrt(head_size);

  VLOG(4) << "FlashAttn fwd dims q[" << q.dims() << "], k[" << k.dims()
          << "], v[" << v.dims() << "]";

  if (is_test) dropout = 0.0f;

  ctx.template Alloc<T>(out);

  hipStream_t stream = ctx.stream();
  const bool is_bf16 = q.dtype() == DataType::BFLOAT16 ? true : false;

  uint64_t seed;
  uint64_t offset;

  if (fixed_seed_offset.get_ptr()) {
    const int64_t* fixed_seed_offset_data =
        fixed_seed_offset.get_ptr()->data<int64_t>();
    seed = static_cast<uint64_t>(fixed_seed_offset_data[0]);
    offset = static_cast<uint64_t>(fixed_seed_offset_data[1]);
  } else {
    uint64_t inc = batch_size * num_heads * 32;
    std::pair<uint64_t, uint64_t> seed_offset_pair;
    if (rng_name != "") {
      auto gen = phi::GetRandomSeedGenerator(rng_name);
      seed_offset_pair = gen->IncrementOffset(inc);
    } else {
      auto* gen = ctx.GetGenerator();
      seed_offset_pair = gen->IncrementOffset(inc);
    }
    seed = seed_offset_pair.first;
    offset = seed_offset_pair.second;
  }

  VLOG(4) << "FlashAttn fwd seed: " << seed << ", offset: " << offset;

  seed_offset->Resize({2});
  int64_t* seed_offset_data = ctx.template HostAlloc<int64_t>(seed_offset);
  seed_offset_data[0] = static_cast<int64_t>(seed);
  seed_offset_data[1] = static_cast<int64_t>(offset);

  auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };

  const int head_size_rounded = round_multiple(head_size, 32);
  const int seqlen_q_rounded = round_multiple(seqlen_q, 128);
  const int seqlen_k_rounded = round_multiple(seqlen_k, 128);

  softmax_lse->Resize({batch_size, num_heads, seqlen_q_rounded});
  ctx.template Alloc<float>(softmax_lse);

  if (return_softmax) {
    softmax->Resize(
        {batch_size, num_heads, seqlen_q_rounded, seqlen_k_rounded});
    ctx.template Alloc<T>(softmax);
  }

  bool succ =
      phi::dynload::flash_attn_fwd(q.data(),
                                   k.data(),
                                   v.data(),
                                   out->data(),
                                   return_softmax ? softmax->data() : nullptr,
                                   softmax_lse->data(),
                                   batch_size,
                                   seqlen_q,
                                   seqlen_k,
                                   seqlen_q_rounded,
                                   seqlen_k_rounded,
                                   num_heads,
                                   num_heads_k,
                                   head_size,
                                   head_size_rounded,
                                   dropout,
                                   scale,
                                   causal,
                                   return_softmax,
                                   is_bf16,
                                   stream,
                                   seed,
                                   offset);

  if (!succ) {
    PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
  }
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(
      phi::Backend::ALL_BACKEND);  // fixed_seed_offset
}

PD_REGISTER_KERNEL(flash_attn,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(3).SetBackend(
      phi::Backend::ALL_BACKEND);  // fixed_seed_offset
}
