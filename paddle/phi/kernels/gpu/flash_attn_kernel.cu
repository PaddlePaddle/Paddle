#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_kernel.h"

#include "glog/logging.h"  // For VLOG()
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/reshape_kernel.h"

#ifdef PADDLE_WITH_FLASHATTN
#include "paddle/phi/backends/dynload/flashattn.h"
#include "paddle/phi/kernels/gpu/flash_attn_utils.h"
#endif

DECLARE_bool(cudnn_deterministic);

namespace phi {

template <typename T>
__global__ void SimleScaleWithMaskKernel(int64_t numel,
                                         float scale,
                                         const T* input,
                                         T* ouput) {
  CUDA_KERNEL_LOOP_TYPE(i, numel, int64_t) {
    ouput[i] = static_cast<T>(scale * static_cast<float>(input[i]));
  }
}

template <typename T, typename Context>
void ComputeScaleQ(
    const Context& ctx, int64_t numel, float scale, const T* input, T* output) {
  auto gpu_config = phi::backends::gpu::GetGpuLaunchConfig1D(ctx, numel, 1);
  SimleScaleWithMaskKernel<<<gpu_config.block_per_grid,
                             gpu_config.thread_per_block,
                             0,
                             ctx.stream()>>>(numel, scale, input, output);
}

template <typename T, typename Context>
void FlashAttnWithMaskUnpaddedImpl(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& cu_seqlens_q,
    const DenseTensor& cu_seqlens_k,
    const paddle::optional<DenseTensor>& fixed_seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    int64_t max_seqlen_q,
    int64_t max_seqlen_k,
    float scale,
    float dropout,
    bool causal,
    bool return_softmax,
    bool is_test,
    const std::string& rng_name,
    DenseTensor* out,
    DenseTensor* softmax,
    DenseTensor* softmax_lse,
    DenseTensor* seed_offset) {
  hipStream_t stream = ctx.stream();

  auto dims = q.dims();
  int64_t total_q = dims[0];
  int64_t num_heads = dims[1];
  int64_t head_size = dims[2];

  int64_t total_k = k.dims()[0];
  int64_t batch_size = cu_seqlens_q.numel() - 1;

  int num_splits = 0;  // 0 for an internal heuristic, which is optimal
  if (FLAGS_cudnn_deterministic) {
    num_splits = 1;
  }
  bool zero_tensors = false;

  uint64_t seed;
  uint64_t offset;

  if (fixed_seed_offset.get_ptr()) {
    const int64_t* fixed_seed_offset_data =
        fixed_seed_offset.get_ptr()->data<int64_t>();
    seed = static_cast<uint64_t>(fixed_seed_offset_data[0]);
    offset = static_cast<uint64_t>(fixed_seed_offset_data[1]);
  } else {
    uint64_t inc = batch_size * num_heads * 32;
    std::pair<uint64_t, uint64_t> seed_offset_pair;
    if (rng_name != "") {
      auto gen = phi::GetRandomSeedGenerator(rng_name);
      seed_offset_pair = gen->IncrementOffset(inc);
    } else {
      auto* gen = ctx.GetGenerator();
      seed_offset_pair = gen->IncrementOffset(inc);
    }
    seed = seed_offset_pair.first;
    offset = seed_offset_pair.second;
  }

  VLOG(4) << "FlashAttn fwd seed: " << seed << ", offset: " << offset
          << ", num_splits:" << num_splits;

  seed_offset->Resize({2});
  int64_t* seed_offset_data = ctx.template HostAlloc<int64_t>(seed_offset);
  seed_offset_data[0] = static_cast<int64_t>(seed);
  seed_offset_data[1] = static_cast<int64_t>(offset);

  int64_t seq_len_q = ((max_seqlen_q + 16 - 1) / 16) * 16;

  softmax_lse->Resize({batch_size, num_heads, seq_len_q});
  ctx.template Alloc<float>(softmax_lse);

  if (return_softmax) {
    // may allocate more space than *max_seqlen_k*
    int64_t blocksize_c = head_size > 64 ? 128 : 256;
    int64_t seq_len_k =
        ((max_seqlen_k + blocksize_c - 1) / blocksize_c) * blocksize_c;
    if (max_seqlen_k <= 128) {
      seq_len_k = 128;
    } else if (max_seqlen_k <= 256) {
      seq_len_k = 256;
    }
    softmax->Resize({batch_size, num_heads, seq_len_q, seq_len_k});
    ctx.template Alloc<T>(softmax);
  }

  uint64_t workspace_size = 0;
  DenseTensor workspace;

  PADDLE_ENFORCE_NE(causal,
                    true,
                    phi::errors::InvalidArgument(
                        "attn_mask is not nullptr, causal can not be true"));
  bool flag = (head_size == 32 || head_size == 64 || head_size == 128);
  PADDLE_ENFORCE_EQ(
      flag,
      true,
      phi::errors::InvalidArgument(
          "Currently, the mask only supports head_dim of 32, 64, and 128"));

  int64_t q_size = total_q * num_heads * head_size;
  DenseTensor scale_q;
  scale_q.Resize({total_q, num_heads, head_size});
  ctx.template Alloc<T>(&scale_q);
  // DenseTensor* scale_q =  new DenseTensor;
  // scale_q->Resize({total_q, num_heads, head_size});
  // ctx.template Alloc<T>(scale_q);
  // compute scale Q
  ComputeScaleQ(ctx, q_size, scale, q.data<T>(), scale_q.data<T>());

  float fa_with_mask_scale = 1.0f;
  std::vector<int64_t> rand_mask_dim;
  const DenseTensor* attn_mask_ptr = attn_mask.get_ptr();
  int64_t first_dim = 1;
  const auto& origin_dims = attn_mask_ptr->dims();
  auto rank = origin_dims.size();
  for (int i = 0; i < rank - 3; i++) {
    first_dim *= origin_dims[i];
  }
  rand_mask_dim = {first_dim,
                   origin_dims[rank - 3],
                   origin_dims[rank - 2],
                   origin_dims[rank - 1]};

  bool succ = phi::dynload::flash_attn_fwd_with_bias_and_mask(
      static_cast<const void*>(scale_q.data()),
      static_cast<const void*>(k.data()),
      static_cast<const void*>(v.data()),
      nullptr,  // for calculation workspace size
      static_cast<const int32_t*>(cu_seqlens_q.data()),
      static_cast<const int32_t*>(cu_seqlens_k.data()),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      max_seqlen_q,
      max_seqlen_k,
      dropout,
      fa_with_mask_scale,
      zero_tensors,
      is_bf16,
      num_splits,
      softmax_lse->data(),
      nullptr,
      &workspace_size,
      stream,
      seed,
      offset,
      attn_mask_ptr ? attn_mask_ptr->data() : nullptr,
      nullptr,
      rand_mask_dim.data() ? rand_mask_dim.data() : nullptr,
      nullptr);
  PADDLE_ENFORCE_EQ(
      succ,
      true,
      phi::errors::External("Error in Flash-Attention, detail information is",
                            phi::dynload::flash_attn_error()));

  if (workspace_size > 0) {
    workspace = Empty<float>(
        ctx, {static_cast<int64_t>(workspace_size / sizeof(float))});
  }
  succ = phi::dynload::flash_attn_fwd_with_bias_and_mask(
      static_cast<const void*>(scale_q.data()),
      k.data(),
      v.data(),
      out->data(),  // set out to nullptr to calculate workspace size
      static_cast<const int32_t*>(cu_seqlens_q.data()),
      static_cast<const int32_t*>(cu_seqlens_k.data()),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      max_seqlen_q,
      max_seqlen_k,
      dropout,
      fa_with_mask_scale,
      zero_tensors,
      is_bf16,
      num_splits,
      softmax_lse->data(),
      workspace_size > 0 ? workspace.data() : nullptr,
      &workspace_size,
      stream,
      seed,
      offset,
      attn_mask_ptr ? attn_mask_ptr->data() : nullptr,
      nullptr,
      rand_mask_dim.data() ? rand_mask_dim.data() : nullptr,
      nullptr);
  PADDLE_ENFORCE_EQ(
      succ,
      true,
      phi::errors::External("Error in Flash-Attention, detail information is",
                            phi::dynload::flash_attn_error()));
}

template <typename T, typename Context>
void FlashAttnUnpaddedKernel(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& cu_seqlens_q,
    const DenseTensor& cu_seqlens_k,
    const paddle::optional<DenseTensor>& fixed_seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    int64_t max_seqlen_q,
    int64_t max_seqlen_k,
    float scale,
    float dropout,
    bool causal,
    bool return_softmax,
    bool is_test,
    const std::string& rng_name,
    DenseTensor* out,
    DenseTensor* softmax,
    DenseTensor* softmax_lse,
    DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(out);

  hipStream_t stream = ctx.stream();

  // q,k,v [total_*, num_heads, head_dim]
  auto dims = q.dims();
  PADDLE_ENFORCE_EQ(
      dims.size(),
      3,
      phi::errors::InvalidArgument("flash_attn_raw receive input with dim "
                                   "[total_seq_len, num_heads, head_dim]"));

  if (attn_mask.get_ptr()) {
    FlashAttnWithMaskUnpaddedImpl<T, Context>(ctx,
                                              q,
                                              k,
                                              v,
                                              cu_seqlens_q,
                                              cu_seqlens_k,
                                              fixed_seed_offset,
                                              attn_mask,
                                              max_seqlen_q,
                                              max_seqlen_k,
                                              scale,
                                              dropout,
                                              causal,
                                              return_softmax,
                                              is_test,
                                              rng_name,
                                              out,
                                              softmax,
                                              softmax_lse,
                                              seed_offset);
  } else {
    const int64_t total_q = dims[0];
    const int64_t num_heads = dims[1];
    const int64_t head_size = dims[2];

    const int64_t total_k = k.dims()[0];
    const int64_t num_heads_k = k.dims()[1];
    const int64_t batch_size = cu_seqlens_q.numel() - 1;

    // TODO(umiswing): add deterministic in fa2.
    // int num_splits = 0;  // 0 for an internal heuristic, which is optimal
    // if (FLAGS_cudnn_deterministic) {
    //   num_splits = 1;
    // }

    // TODO(umiswing): add shape check

    FlashAttnFwdParamsV2<T> params =
        FlashAttnFwdParamsV2<T>(ctx,
                                batch_size,
                                max_seqlen_q,
                                max_seqlen_k,
                                num_heads,
                                num_heads_k,
                                head_size,
                                dropout,
                                scale,
                                causal,
                                return_softmax,
                                q.dtype(),
                                is_test,
                                rng_name,
                                fixed_seed_offset.get_ptr(),
                                softmax,
                                softmax_lse,
                                seed_offset);

    VLOG(4) << "FlashAttn fwd seed: " << params.seed
            << ", offset: " << params.offset;

    const bool succ = phi::dynload::flash_attn_varlen_fwd(
        q.data(),
        k.data(),
        v.data(),
        cu_seqlens_q.data<int32_t>(),
        cu_seqlens_k.data<int32_t>(),
        params.rng_state.data(),
        out->data(),
        params.return_softmax ? softmax->data() : nullptr,
        softmax_lse->data(),
        params.batch_size,
        params.max_seqlen_q,
        params.max_seqlen_k,
        params.seqlen_q_rounded,
        params.seqlen_k_rounded,
        params.num_heads,
        params.num_heads_k,
        params.head_size,
        params.head_size_rounded,
        params.dropout,
        params.scale,
        params.causal,
        params.return_softmax,
        params.is_bf16,
        stream,
        params.seed,
        params.offset);
    if (!succ) {
      PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
    }
  }
#else
  PADDLE_THROW(
      phi::errors::Unimplemented("FlashAttention is unsupported, please check "
                                 "the GPU compability and CUDA Version."));
#endif
}

template <typename T, typename Context>
void FlashAttnKernel(const Context& ctx,
                     const DenseTensor& q,
                     const DenseTensor& k,
                     const DenseTensor& v,
                     const paddle::optional<DenseTensor>& fixed_seed_offset,
                     const paddle::optional<DenseTensor>& attn_mask,
                     float dropout,
                     bool causal,
                     bool return_softmax,
                     bool is_test,
                     const std::string& rng_name,
                     DenseTensor* out,
                     DenseTensor* softmax,
                     DenseTensor* softmax_lse,
                     DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [batch_size, seq_len, num_heads, head_dim]
  auto dims = q.dims();
  PADDLE_ENFORCE_EQ(dims.size(),
                    4,
                    phi::errors::InvalidArgument(
                        "flash_attn receive input with dim "
                        "[batch_size, seq_len, num_heads, head_dim]"));

  const int batch_size = dims[0];
  const int seqlen_q = dims[1];
  const int num_heads = dims[2];
  const int head_size = dims[3];
  const int seqlen_k = k.dims()[1];
  const int num_heads_k = k.dims()[2];

  // TODO(umiswing): Add check shape

  const float scale = 1.0f / std::sqrt(head_size);
  if (!attn_mask.get_ptr()) {
    FlashAttnFwdParamsV2<T> params =
        FlashAttnFwdParamsV2<T>(ctx,
                                batch_size,
                                seqlen_q,
                                seqlen_k,
                                num_heads,
                                num_heads_k,
                                head_size,
                                dropout,
                                scale,
                                causal,
                                return_softmax,
                                q.dtype(),
                                is_test,
                                rng_name,
                                fixed_seed_offset.get_ptr(),
                                softmax,
                                softmax_lse,
                                seed_offset);

    VLOG(4) << "FlashAttn fwd dims q[" << q.dims() << "], k[" << k.dims()
            << "], v[" << v.dims() << "]";

    ctx.template Alloc<T>(out);

    hipStream_t stream = ctx.stream();

    VLOG(4) << "FlashAttn fwd seed: " << params.seed
            << ", offset: " << params.offset;

    bool succ = phi::dynload::flash_attn_fwd(
        q.data(),
        k.data(),
        v.data(),
        params.rng_state.data(),
        out->data(),
        params.return_softmax ? params.softmax->data() : nullptr,
        params.softmax_lse->data(),
        params.batch_size,
        params.max_seqlen_q,
        params.max_seqlen_k,
        params.seqlen_q_rounded,
        params.seqlen_k_rounded,
        params.num_heads,
        params.num_heads_k,
        params.head_size,
        params.head_size_rounded,
        params.dropout,
        params.scale,
        params.causal,
        params.return_softmax,
        params.is_bf16,
        stream,
        params.seed,
        params.offset);

    PADDLE_ENFORCE_EQ(
        succ,
        true,
        phi::errors::External(
            "Error in Flash-Attention-2, detail information is: %s",
            phi::dynload::flash_attn_error()));
  } else {
    DenseTensor q_t_s, k_t_s, v_t_s;
    q_t_s.ShareDataWith(q).Resize({total_q, num_heads, head_size});
    k_t_s.ShareDataWith(k).Resize({total_k, num_heads, head_size});
    v_t_s.ShareDataWith(v).Resize({total_k, num_heads, head_size});

    DenseTensor cu_seqlens_q;
    DenseTensor cu_seqlens_k;
    ArangeNullaryKernel<int32_t, Context>(
        ctx, 0, (batch_size + 1) * seq_len_q, seq_len_q, &cu_seqlens_q);
    ArangeNullaryKernel<int32_t, Context>(
        ctx, 0, (batch_size + 1) * seq_len_k, seq_len_k, &cu_seqlens_k);

    FlashAttnUnpaddedKernel<T, Context>(ctx,
                                        q_t_s,
                                        k_t_s,
                                        v_t_s,
                                        cu_seqlens_q,
                                        cu_seqlens_k,
                                        fixed_seed_offset,
                                        attn_mask,
                                        seq_len_q,
                                        seq_len_k,
                                        scale,
                                        dropout,
                                        causal,
                                        return_softmax,
                                        is_test,
                                        rng_name,
                                        out,
                                        softmax,
                                        softmax_lse,
                                        seed_offset);
  }
#else
  PADDLE_THROW(phi::errors::Unimplemented(
      "FlashAttention is unsupported, please set use_flash_attn to false."));
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(
      phi::Backend::ALL_BACKEND);  // fixed_seed_offset
}

PD_REGISTER_KERNEL(flash_attn,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(3).SetBackend(
      phi::Backend::ALL_BACKEND);  // fixed_seed_offset
}
