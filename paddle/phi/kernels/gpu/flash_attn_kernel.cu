#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_kernel.h"

#include "glog/logging.h"  // For VLOG()
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/gpu/flash_attn_utils.h"
#include "paddle/phi/kernels/reshape_kernel.h"

PD_DECLARE_bool(cudnn_deterministic);

namespace phi {

template <typename T, typename Context>
void FlashAttnUnpaddedKernel(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& cu_seqlens_q,
    const DenseTensor& cu_seqlens_k,
    const paddle::optional<DenseTensor>& fixed_seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    int64_t max_seqlen_q,
    int64_t max_seqlen_k,
    float scale,
    float dropout,
    bool causal,
    bool return_softmax,
    bool is_test,
    const std::string& rng_name,
    DenseTensor* out,
    DenseTensor* softmax,
    DenseTensor* softmax_lse,
    DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(out);

  hipStream_t stream = ctx.stream();

  // q,k,v [total_*, num_heads, head_dim]
  auto dims = q.dims();
  PADDLE_ENFORCE_EQ(
      dims.size(),
      3,
      phi::errors::InvalidArgument("flash_attn_raw receive input with dim "
                                   "[total_seq_len, num_heads, head_dim]"));

  const int64_t total_q = dims[0];
  const int64_t num_heads = dims[1];
  const int64_t head_size = dims[2];

  const int64_t total_k = k.dims()[0];
  const int64_t num_heads_k = k.dims()[1];
  const int64_t batch_size = cu_seqlens_q.numel() - 1;

  // TODO(umiswing): add deterministic in fa2.
  // int num_splits = 0;  // 0 for an internal heuristic, which is optimal
  // if (FLAGS_cudnn_deterministic) {
  //   num_splits = 1;
  // }

  // TODO(umiswing): add shape check

  const DenseTensor* attn_mask_tensor = attn_mask.get_ptr();
  std::vector<int64_t> mask_dims = GetAttnMaskDims(attn_mask_tensor);

  FlashAttnFwdParamsV2<T> params = FlashAttnFwdParamsV2<T>(ctx,
                                                           batch_size,
                                                           max_seqlen_q,
                                                           max_seqlen_k,
                                                           num_heads,
                                                           num_heads_k,
                                                           head_size,
                                                           dropout,
                                                           scale,
                                                           causal,
                                                           return_softmax,
                                                           q.dtype(),
                                                           is_test,
                                                           rng_name,
                                                           fixed_seed_offset,
                                                           softmax,
                                                           softmax_lse,
                                                           seed_offset);

  VLOG(10) << "FlashAttn fwd seed: " << params.seed
           << ", offset: " << params.offset;

  bool succ = phi::dynload::flash_attn_varlen_fwd(
      q.data(),
      k.data(),
      v.data(),
      cu_seqlens_q.data<int32_t>(),
      cu_seqlens_k.data<int32_t>(),
      params.rng_state.data(),
      out->data(),
      params.return_softmax ? softmax->data() : nullptr,
      softmax_lse->data(),
      params.batch_size,
      params.max_seqlen_q,
      params.max_seqlen_k,
      params.seqlen_q_rounded,
      params.seqlen_k_rounded,
      params.num_heads,
      params.num_heads_k,
      params.head_size,
      params.head_size_rounded,
      params.dropout,
      params.scale,
      params.causal,
      params.return_softmax,
      params.is_bf16,
      stream,
      params.seed,
      params.offset,
      attn_mask_tensor ? attn_mask_tensor->data() : nullptr,
      mask_dims.data());
  CheckFlashAttnStatus(succ);
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnKernel(const Context& ctx,
                     const DenseTensor& q,
                     const DenseTensor& k,
                     const DenseTensor& v,
                     const paddle::optional<DenseTensor>& fixed_seed_offset,
                     const paddle::optional<DenseTensor>& attn_mask,
                     float dropout,
                     bool causal,
                     bool return_softmax,
                     bool is_test,
                     const std::string& rng_name,
                     DenseTensor* out,
                     DenseTensor* softmax,
                     DenseTensor* softmax_lse,
                     DenseTensor* seed_offset) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [batch_size, seq_len, num_heads, head_dim]
  const auto& dims = q.dims();
  PADDLE_ENFORCE_EQ(dims.size(),
                    4,
                    phi::errors::InvalidArgument(
                        "flash_attn receive input with dim "
                        "[batch_size, seq_len, num_heads, head_dim]"));

  const int64_t batch_size = dims[0];
  const int64_t seqlen_q = dims[1];
  const int64_t num_heads = dims[2];
  const int64_t head_size = dims[3];
  const int64_t seqlen_k = k.dims()[1];
  const int64_t num_heads_k = k.dims()[2];

  const int64_t total_q = batch_size * seqlen_q;
  const int64_t total_k = batch_size * seqlen_k;

  // TODO(umiswing): Add check shape

  const float scale = 1.0f / std::sqrt(head_size);

  const DenseTensor* attn_mask_tensor = attn_mask.get_ptr();
  std::vector<int64_t> mask_dims = GetAttnMaskDims(attn_mask_tensor);

  FlashAttnFwdParamsV2<T> params = FlashAttnFwdParamsV2<T>(ctx,
                                                           batch_size,
                                                           seqlen_q,
                                                           seqlen_k,
                                                           num_heads,
                                                           num_heads_k,
                                                           head_size,
                                                           dropout,
                                                           scale,
                                                           causal,
                                                           return_softmax,
                                                           q.dtype(),
                                                           is_test,
                                                           rng_name,
                                                           fixed_seed_offset,
                                                           softmax,
                                                           softmax_lse,
                                                           seed_offset);

  VLOG(10) << "FlashAttn fwd dims: q[" << q.dims() << "], k[" << k.dims()
           << "], v[" << v.dims() << "]";
  VLOG(10) << "FlashAttn fwd seed: " << params.seed
           << ", offset: " << params.offset;

  ctx.template Alloc<T>(out);

  hipStream_t stream = ctx.stream();

  int64_t q_size = batch_size * seqlen_q * num_heads * head_size;
  DenseTensor scaled_q =
      Empty<T>(ctx, {batch_size, seqlen_q, num_heads, head_size});
  ComputeScaleQ(ctx, q_size, scale, q.data<T>(), scaled_q.data<T>());

  bool succ = phi::dynload::flash_attn_fwd(
      // attn_mask_tensor ? scaled_q.data() : q.data(),
      q.data(),
      k.data(),
      v.data(),
      params.rng_state.data(),
      out->data(),
      params.return_softmax ? params.softmax->data() : nullptr,
      params.softmax_lse->data(),
      params.batch_size,
      params.max_seqlen_q,
      params.max_seqlen_k,
      params.seqlen_q_rounded,
      params.seqlen_k_rounded,
      params.num_heads,
      params.num_heads_k,
      params.head_size,
      params.head_size_rounded,
      params.dropout,
      // attn_mask_tensor ? 1.0f : params.scale,
      params.scale,
      std::sqrt(head_size),  // for unscale
      params.causal,
      params.return_softmax,
      params.is_bf16,
      stream,
      params.seed,
      params.offset,
      attn_mask_tensor ? attn_mask_tensor->data() : nullptr,
      mask_dims.data());
  CheckFlashAttnStatus(succ);
#else
  RaiseNotSupportedError();
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(
      phi::Backend::ALL_BACKEND);  // fixed_seed_offset
}

PD_REGISTER_KERNEL(flash_attn,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(3).SetBackend(
      phi::Backend::ALL_BACKEND);  // fixed_seed_offset
}
