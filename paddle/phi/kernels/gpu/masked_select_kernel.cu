#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/scan.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/masked_select_kernel.h"

namespace phi {

__global__ void SetMaskArray(const bool* mask, int32_t* mask_array, int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx])
      mask_array[idx] = 1;
    else
      mask_array[idx] = 0;
  }
}

template <typename T>
__global__ void SelectWithPrefixMask(const int32_t* mask_prefix_sum,
                                     const bool* mask,
                                     const T* input,
                                     T* out,
                                     int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx]) {
      int index = mask_prefix_sum[idx];
      out[index] = input[idx];
    }
  }
}

template <typename T, typename Context>
void MaskedSelectKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& mask,
                        DenseTensor* out) {
  auto* mask_data = mask.data<bool>();
  auto input_data = x.data<T>();

  auto mask_size = mask.numel();
  auto input_dim = x.dims();
  auto mask_dim = mask.dims();
  PADDLE_ENFORCE_EQ(input_dim,
                    mask_dim,
                    phi::errors::InvalidArgument(
                        "The dim size of input and mask in OP(masked_selected) "
                        "must be equal, but got input dim:(%ld), mask dim: "
                        "(%ld). Please check input "
                        "value.",
                        input_dim,
                        mask_dim));

  thrust::device_ptr<const bool> mask_dev_ptr =
      thrust::device_pointer_cast(mask_data);
  thrust::device_vector<T> mask_vec(mask_dev_ptr, mask_dev_ptr + mask_size);
  auto out_size = thrust::count(mask_vec.begin(), mask_vec.end(), true);

  DDim out_dim{out_size};
  out->Resize(out_dim);
  auto out_data = out->mutable_data<T>(dev_ctx.GetPlace());

  DenseTensor mask_array;
  DenseTensor mask_prefix_sum;
  mask_array.Resize(mask_dim);
  mask_prefix_sum.Resize(mask_dim);

  int32_t* mask_array_data =
      mask_array.mutable_data<int32_t>(dev_ctx.GetPlace());
  int32_t* mask_prefix_sum_data =
      mask_prefix_sum.mutable_data<int32_t>(dev_ctx.GetPlace());
  int threads = 512;
  int grid = (mask_size + threads - 1) / threads;
  auto stream = dev_ctx.stream();
  SetMaskArray<<<grid, threads, 0, stream>>>(
      mask_data, mask_array_data, mask_size);

  thrust::device_ptr<int32_t> mask_array_dev_ptr =
      thrust::device_pointer_cast(mask_array_data);
  thrust::device_vector<int32_t> mask_array_vec(mask_array_dev_ptr,
                                                mask_array_dev_ptr + mask_size);
  thrust::exclusive_scan(thrust::device,
                         mask_array_vec.begin(),
                         mask_array_vec.end(),
                         mask_prefix_sum_data);

  SelectWithPrefixMask<T><<<grid, threads, 0, stream>>>(
      mask_prefix_sum_data, mask_data, input_data, out_data, mask_size);
}

}  // namespace phi

PD_REGISTER_KERNEL(masked_select,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaskedSelectKernel,
                   float,
                   double,
                   int,
                   int64_t) {
  kernel->InputAt(1).SetDataType(phi::DataType::BOOL);
}
