// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/masked_select_kernel.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/scan.h>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/select_impl.cu.h"

namespace phi {

template <typename MT, typename InT, typename OutT>
struct MaskedSelectFunctor {
  HOSTDEVICE MaskedSelectFunctor() {}

  HOSTDEVICE inline void operator()(OutT* out,
                                    const MT* mask,
                                    const InT* value,
                                    int num) {
    int store_fix = 0;
    for (int idx = 0; idx < num; idx++) {
      if (mask[idx]) {
        out[store_fix++] = value[idx];
      }
    }
  }
};

template <typename T, typename Context>
void MaskedSelectKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& mask,
                        DenseTensor* out) {
  auto* mask_data = mask.data<bool>();
  auto input_data = x.data<T>();

  auto mask_size = mask.numel();
  auto input_dim = x.dims();
  auto mask_dim = mask.dims();
  PADDLE_ENFORCE_EQ(input_dim,
                    mask_dim,
                    phi::errors::InvalidArgument(
                        "The dim size of input and mask in OP(masked_selected) "
                        "must be equal, but got input dim:(%ld), mask dim: "
                        "(%ld). Please check input "
                        "value.",
                        input_dim,
                        mask_dim));
  using Functor = MaskedSelectFunctor<bool, T, T>;
  phi::funcs::SelectKernel<bool, T, T, 1, Functor>(
      dev_ctx, mask, x, out, Functor());
}

}  // namespace phi

PD_REGISTER_KERNEL(masked_select,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaskedSelectKernel,
                   float,
                   double,
                   int,
                   int64_t) {
  kernel->InputAt(1).SetDataType(phi::DataType::BOOL);
}
