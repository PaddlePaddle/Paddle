#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/embedding_grad_kernel.h"

#include "paddle/fluid/framework/mixed_vector.h"
#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/embedding_util.h"

namespace phi {

template <typename InT, typename OutT>
__global__ void InputTypeConvert(const InT* in_ids,
                                 const int64_t K,
                                 OutT* out_ids) {
  for (int i = 0; i < K; i++) {
    out_ids[i] = static_cast<OutT>(in_ids[i]);
  }
}

template <typename T, typename IdT>
__global__ void EmbeddingGrad(T* table,
                              const T* output,
                              const IdT* ids,
                              const int64_t N,
                              const int64_t K,
                              const int64_t D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * gridDim.x;

  while (idy < K) {
    auto id = static_cast<int64_t>(ids[idy]);
    const T* out = output + idy * D;
    T* tab = table + id * D;
#ifdef PADDLE_WITH_CUDA
    paddle::platform::VectorizedAtomicAddPerBlock(D, idx, blockDim.x, out, tab);
#else
    for (int i = idx; i < D; i += blockDim.x) {
      paddle::platform::CudaAtomicAdd(&tab[i], out[i]);
    }
#endif
    idy += blockDim.y * gridDim.x;
  }
}

template <typename T, typename Context>
struct EmbeddingGradCUDAFunctor {
  EmbeddingGradCUDAFunctor(const Context& dev_ctx,
                           const DenseTensor& input,
                           const DenseTensor& weight,
                           const DenseTensor& out_grad,
                           int64_t padding_idx,
                           DenseTensor* weight_grad)
      : dev_ctx_(dev_ctx),
        input_(input),
        weight_(weight),
        out_grad_(out_grad),
        padding_idx_(padding_idx),
        weight_grad_(weight_grad) {}

  template <typename IdT>
  void apply() {
    // Since paddings are not trainable and fixed in forward, the gradient of
    // paddings makes no sense and we don't deal with it in backward.
    {
      auto d_output_t = out_grad_;
      auto d_table_t = weight_grad_;

      int N = weight_grad_->dims()[0];
      int D = weight_grad_->dims()[1];
      int K = input_.numel();

      const T* d_output = d_output_t.template data<T>();
      const auto* ids = input_.template data<IdT>();
      T* d_table = dev_ctx_.template Alloc<T>(d_table_t);

#ifdef PADDLE_WITH_HIP
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemsetAsync(d_table, 0, N * D * sizeof(T), dev_ctx_.stream()));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(
          hipMemsetAsync(d_table, 0, N * D * sizeof(T), dev_ctx_.stream()));
#endif

      const int gridx = 2 * dev_ctx_.GetSMCount();
      dim3 threads(128, 8);
      dim3 grids(gridx, 1);
      EmbeddingGrad<T, IdT><<<grids, threads, 0, dev_ctx_.stream()>>>(
          d_table, d_output, ids, N, K, D);
    }
  }

 private:
  const phi::GPUContext& dev_ctx_;
  const DenseTensor& input_;
  const DenseTensor& weight_;
  const DenseTensor& out_grad_;
  int64_t padding_idx_;
  DenseTensor* weight_grad_;
};

template <typename T, typename Context>
void EmbeddingGradKernel(const Context& ctx,
                         const DenseTensor& input,
                         const DenseTensor& weight,
                         const DenseTensor& out_grad,
                         int64_t padding_idx,
                         DenseTensor* weight_grad) {
  EmbeddingGradCUDAFunctor<T, Context> functor(
      ctx, input, weight, out_grad, padding_idx, weight_grad);

  if (input.dtype() == phi::DataType::INT32) {
    functor.template apply<int>();
  } else if (input.dtype() == phi::DataType::INT64) {
    functor.template apply<int64_t>();
  } else if (input.dtype() == phi::DataType::INT16) {
    functor.template apply<int16_t>();
  } else {
    PADDLE_THROW(phi::errors::Unimplemented(
        "emebdding input only support int16, int32 and int64"));
  }
}

template <typename T, typename Context>
struct EmbeddingSparseGradCUDAFunctor {
  EmbeddingSparseGradCUDAFunctor(const Context& dev_ctx,
                                 const DenseTensor& input,
                                 const DenseTensor& weight,
                                 const DenseTensor& out_grad,
                                 int64_t padding_idx,
                                 SelectedRows* weight_grad)
      : dev_ctx_(dev_ctx),
        input_(input),
        weight_(weight),
        out_grad_(out_grad),
        padding_idx_(padding_idx),
        weight_grad_(weight_grad) {}

  template <typename IdT>
  void apply() {
    // Since paddings are not trainable and fixed in forward, the gradient of
    // paddings makes no sense and we don't deal with it in backward.

    const auto* ids_data = input_.template data<IdT>();
    auto* d_table = weight_grad_;
    auto* table = &weight_;
    auto* d_output = &out_grad_;
    int64_t ids_num = input_.numel();
    dim3 threads(128, 8);
    dim3 grids(8, 1);
    auto stream = dev_ctx_.stream();
    paddle::framework::Vector<int64_t> new_rows;
    new_rows.resize(ids_num);
    auto gpu_place = dev_ctx_.GetPlace();

    paddle::framework::MixVector<int64_t> mixv_new_rows(&new_rows);
    if (!std::is_same<IdT, int64_t>::value) {
      InputTypeConvert<<<grids, threads, 0, stream>>>(
          ids_data, ids_num, mixv_new_rows.MutableData(gpu_place));
    } else {
      paddle::memory::Copy(gpu_place,
                           mixv_new_rows.CUDAMutableData(gpu_place),
                           gpu_place,
                           ids_data,
                           ids_num * sizeof(int64_t),
                           stream);
    }

    mixv_new_rows.CopyToCPU();
    d_table->set_rows(new_rows);

    auto* d_table_value = d_table->mutable_value();
    d_table_value->Resize({ids_num, table->dims()[1]});
    dev_ctx_.template Alloc<T>(d_table_value);

    auto* d_table_data = d_table_value->template data<T>();
    auto* d_output_data = d_output->template data<T>();
    auto d_output_dims = d_output->dims();
    auto d_output_dims_2d =
        phi::flatten_to_2d(d_output_dims, d_output_dims.size() - 1);
    PADDLE_ENFORCE_EQ(d_table_value->dims(),
                      d_output_dims_2d,
                      phi::errors::InvalidArgument(
                          "ShapeError: The shape of lookup_table@Grad and "
                          "output@Grad should be same. "
                          "But received lookup_table@Grad's shape = [%s], "
                          "output@Grad's shape = [%s].",
                          d_table_value->dims(),
                          d_output_dims_2d));
    paddle::memory::Copy(gpu_place,
                         d_table_data,
                         gpu_place,
                         d_output_data,
                         d_output->numel() * sizeof(T),
                         stream);
  }

 private:
  const phi::GPUContext& dev_ctx_;
  const DenseTensor& input_;
  const DenseTensor& weight_;
  const DenseTensor& out_grad_;
  int64_t padding_idx_;
  SelectedRows* weight_grad_;
};

template <typename T, typename Context>
void EmbeddingSparseGradKernel(const Context& ctx,
                               const DenseTensor& input,
                               const DenseTensor& weight,
                               const DenseTensor& out_grad,
                               int64_t padding_idx,
                               SelectedRows* weight_grad) {
  EmbeddingSparseGradCUDAFunctor<T, Context> functor(
      ctx, input, weight, out_grad, padding_idx, weight_grad);

  if (input.dtype() == phi::DataType::INT32) {
    functor.template apply<int>();
  } else if (input.dtype() == phi::DataType::INT64) {
    functor.template apply<int64_t>();
  } else if (input.dtype() == phi::DataType::INT16) {
    functor.template apply<int16_t>();
    PADDLE_THROW(phi::errors::Unimplemented(
        "emebdding input only support int16, int32 and int64"));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(embedding_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::EmbeddingGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(embedding_sparse_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::EmbeddingSparseGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
