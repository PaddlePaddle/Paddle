#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/trunc_grad_kernel.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename T>
__global__ void TruncGrad(T* dx, int64_t N) {
  CUDA_KERNEL_LOOP(index, N) { dx[index] = static_cast<T>(0.0); }
}

template <typename T, typename Context>
void TruncGradKernel(const Context& dev_ctx,
                     const DenseTensor& out_grad,
                     DenseTensor* in_grad) {
  const auto* out_grad_data = out_grad.data<T>();
  T* in_grad_data = dev_ctx.template Alloc<T>(in_grad);

  int64_t numel = out_grad.numel();

  int theads = PADDLE_CUDA_NUM_THREADS;
  int blocks = (numel + theads - 1) / theads;

  TruncGrad<<<blocks, theads>>>(in_grad_data, numel);
}

}  // namespace phi

PD_REGISTER_KERNEL(trunc_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::TruncGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
