// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/put_along_axis_kernel.h"

#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/operators/gather_scatter_kernel.h"
#include "paddle/fluid/platform/place.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"

namespace phi {

template <typename T, typename Context>
void PutAlongAxisKernel(const Context& dev_ctx,
                        const DenseTensor& x,
                        const DenseTensor& index,
                        const DenseTensor& value,
                        int axis,
                        const std::string& reduce,
                        DenseTensor* out) {
  PADDLE_ENFORCE_EQ(paddle::platform::is_gpu_place(dev_ctx.GetPlace()),
                    true,
                    errors::PreconditionNotMet(
                        "PutAlongAxisCUDAKernel only runs on GPU device."));

  const auto& index_type =
      paddle::framework::TransToProtoVarType(index.dtype());

  phi::Copy(dev_ctx, x, dev_ctx.GetPlace(), false, out);
  if (reduce == "add") {
    if (index_type == paddle::framework::proto::VarType::INT32) {
      paddle::operators::gpu_scatter_add_kernel<T, int32_t>(
          *out, axis, index, value, dev_ctx);
    } else if (index_type == paddle::framework::proto::VarType::INT64) {
      paddle::operators::gpu_scatter_add_kernel<T, int64_t>(
          *out, axis, index, value, dev_ctx);
    }
  } else if (reduce == "multiply" || reduce == "mul") {
    if (index_type == paddle::framework::proto::VarType::INT32) {
      paddle::operators::gpu_scatter_mul_kernel<T, int32_t>(
          *out, axis, index, value, dev_ctx);
    } else if (index_type == paddle::framework::proto::VarType::INT64) {
      paddle::operators::gpu_scatter_mul_kernel<T, int64_t>(
          *out, axis, index, value, dev_ctx);
    }
  } else if (reduce == "assign") {
    if (index_type == paddle::framework::proto::VarType::INT32) {
      paddle::operators::gpu_scatter_assign_kernel<T, int32_t>(
          *out, axis, index, value, dev_ctx);
    } else if (index_type == paddle::framework::proto::VarType::INT64) {
      paddle::operators::gpu_scatter_assign_kernel<T, int64_t>(
          *out, axis, index, value, dev_ctx);
    }
  } else {
    PADDLE_THROW(errors::InvalidArgument(
        "can not support reduce: '%s' for scatter kernel, only "
        "support reduce op: 'add', 'assign', 'mul' and 'multiply', the "
        "default reduce op is 'assign' ",
        reduce));
    return;
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(put_along_axis,
                   GPU,
                   ALL_LAYOUT,
                   phi::PutAlongAxisKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   phi::dtype::float16) {}
