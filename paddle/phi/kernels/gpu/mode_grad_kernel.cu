#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/mode_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/mode.h"

namespace phi {

template <typename T>
__global__ void AssignGradWithAxis(const T* grad_out,
                                   const int64_t* indices,
                                   T* grad_in,
                                   int pre,
                                   int post,
                                   int raw_height,
                                   int k) {
  // raw_height is the length of topk axis
  for (int i = blockIdx.x; i < pre; i += gridDim.x) {
    int base_index = i * post * k;
    int base_grad = i * post * raw_height;
    for (int j = threadIdx.x; j < raw_height * post; j += blockDim.x) {
      grad_in[base_grad + j] = static_cast<T>(0);
    }
    __syncthreads();
    for (int j = threadIdx.x; j < k * post; j += blockDim.x) {
      int64_t idx_ij = indices[base_index + j];
      int64_t in_ij = base_grad + (idx_ij * post) + (j % post);
      grad_in[in_ij] = grad_out[base_index + j];
    }
  }
}

template <typename T, typename Context>
void ModeGradKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& indices,
                    const DenseTensor& out_grad,
                    int axis,
                    bool keepdim,
                    DenseTensor* x_grad) {
  const auto& in_dims = x.dims();
  auto out_dims = indices.dims();

  if (axis < 0) axis += in_dims.size();
  // allocate the cuda memory for the x_grad
  T* x_grad_data = dev_ctx.template Alloc<T>(x_grad);
  const T* out_grad_data = out_grad.data<T>();
  const int64_t* indices_data = indices.data<int64_t>();

  int pre, n, post;
  funcs::GetDims(in_dims, axis, &pre, &n, &post);

  // calcluate the block and grid num
  int block_size = funcs::ComputeBlockSize(post);
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
  int grid_size = std::min(max_blocks, pre);
  AssignGradWithAxis<T><<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
      out_grad_data, indices_data, x_grad_data, pre, post, n, 1);
}

}  // namespace phi

PD_REGISTER_KERNEL(mode_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::ModeGradKernel,
                   float,
                   double,
                   int32_t,
                   int64_t) {}
