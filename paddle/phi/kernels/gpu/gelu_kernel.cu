#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// clang-format will try to sort headers according to google c++ style,
// and that cause compiling problems.
// clang-format off
#include "paddle/phi/kernels/gelu_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/broadcast_function.h"
#include "paddle/phi/kernels/gpu/gelu_funcs.h"
// clang-format on

DECLARE_bool(use_fast_math);

namespace phi {

template <typename T>
struct GeluWithApproximateFunctor {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // this function is tanh approximation of gelu
    MPType x = static_cast<MPType>(arg_x);
    MPType one = static_cast<MPType>(1);
    MPType half = static_cast<MPType>(0.5);
    MPType kAlpha = static_cast<MPType>(M_2_SQRTPI * M_SQRT1_2);
    auto tanh_out =
        tanh(kAlpha * x * (one + static_cast<MPType>(GELU_CONSTANT) * x * x));
    MPType out = x * half * (one + tanh_out);
    return static_cast<T>(out);
  }
};

template <typename T>
struct GeluWithoutApproximateFunctor {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  inline HOSTDEVICE T operator()(T arg_x) {
    // actual gelu with approximation = false
    MPType x = static_cast<MPType>(arg_x);
    return static_cast<T>(x * normcdf(x));
  }
};

template <typename T, typename Context>
void GeluKernel(const Context& dev_ctx,
                const DenseTensor& x,
                bool approximate,
                DenseTensor* out) {
  dev_ctx.template Alloc<T>(out);
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  if (approximate) {
#ifdef __NVCC__
    if (std::is_same<T, dtype::float16>::value) {
      size_t n = x.numel();
      const auto* in_ptr = reinterpret_cast<const __half*>(x.data<T>());
      auto* out_ptr = reinterpret_cast<__half*>(out->data<T>());
      if (TryLaunchFP16FastGeluFwdVectorizeCUDAKernel(
              dev_ctx, in_ptr, out_ptr, n)) {
        return;
      }
    }
#endif
    using Functor = GeluWithApproximateFunctor<T>;
    phi::funcs::ElementwiseKernel<T, Functor, 1>(
        dev_ctx, ins, &outs, Functor());
  } else {
    using Functor = GeluWithoutApproximateFunctor<T>;
    phi::funcs::ElementwiseKernel<T, Functor, 1>(
        dev_ctx, ins, &outs, Functor());
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(gelu,
                   GPU,
                   ALL_LAYOUT,
                   phi::GeluKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
