#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/radam_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"

namespace phi {
template <typename T, typename MT>
__global__ void RAdamGPUKernel(const T* param,
                               const T* grad,
                               const MT* learning_rate,
                               const MT* moment1,
                               const MT* moment2,
                               const MT* master_param,
                               MT beta1,
                               MT beta2,
                               MT epsilon,
                               MT beta1_pow_scalar,
                               MT beta2_pow_scalar,
                               MT rho_scalar,
                               MT rho_inf,
                               int num,
                               T* param_out,
                               MT* moment1_out,
                               MT* moment2_out,
                               MT* master_param_out) {
  MT lr_scalar = static_cast<MT>(learning_rate[0]);

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int index = idx; index < num; index += gridDim.x * blockDim.x) {
    // load and cast input to MT
    MT d_param =
        master_param ? master_param[index] : static_cast<MT>(param[index]);
    MT d_grad = static_cast<MT>(grad[index]);
    MT d_moment1 = static_cast<MT>(moment1[index]);
    MT d_moment2 = static_cast<MT>(moment2[index]);
    // compute
    MT m1_out = beta1 * d_moment1 + (static_cast<MT>(1) - beta1) * d_grad;
    MT m2_out =
        beta2 * d_moment2 + (static_cast<MT>(1) - beta2) * d_grad * d_grad;

    MT m1_hat = m1_out / (static_cast<MT>(1) - beta1_pow_scalar);

    MT rho_t = rho_inf - static_cast<MT>(2) * rho_scalar;
    MT p_out = static_cast<MT>(0);

    if (rho_t > static_cast<MT>(5)) {
      MT l_t = std::sqrt((static_cast<MT>(1) - beta2_pow_scalar)) /
               (std::sqrt(m2_out) + epsilon);
      MT r_t = std::sqrt(((rho_t - static_cast<MT>(4)) *
                          (rho_t - static_cast<MT>(2)) * rho_inf) /
                         ((rho_inf - static_cast<MT>(4)) *
                          (rho_inf - static_cast<MT>(2)) * rho_t));

      p_out = d_param - lr_scalar * m1_hat * r_t * l_t;
    } else {
      p_out = d_param - lr_scalar * m1_hat;
    }

    // store
    param_out[index] = static_cast<T>(p_out);
    moment1_out[index] = static_cast<MT>(m1_out);
    moment2_out[index] = static_cast<MT>(m2_out);

    if (master_param_out) {
      master_param_out[index] = p_out;
    }
  }
}

template <typename T, typename Context>
void RAdamKernel(const Context& dev_ctx,
                 const DenseTensor& param,
                 const DenseTensor& grad,
                 const DenseTensor& learning_rate,
                 const DenseTensor& beta1_pow,
                 const DenseTensor& beta2_pow,
                 const DenseTensor& rho,
                 const DenseTensor& moment1,
                 const DenseTensor& moment2,
                 const paddle::optional<DenseTensor>& master_param,
                 float beta1,
                 float beta2,
                 float epsilon,
                 bool multi_precision,
                 DenseTensor* param_out,
                 DenseTensor* beta1_pow_out,
                 DenseTensor* beta2_pow_out,
                 DenseTensor* rho_out,
                 DenseTensor* moment1_out,
                 DenseTensor* moment2_out,
                 DenseTensor* master_param_out) {
  using MPDType = typename phi::dtype::template MPTypeTrait<T>::Type;
  T* param_out_data = dev_ctx.template Alloc<T>(param_out);

  MPDType* beta1_pow_out_data =
      dev_ctx.template HostAlloc<MPDType>(beta1_pow_out);
  MPDType* beta2_pow_out_data =
      dev_ctx.template HostAlloc<MPDType>(beta2_pow_out);
  MPDType* rho_out_data = dev_ctx.template HostAlloc<MPDType>(rho_out);

  MPDType* moment1_out_data = dev_ctx.template Alloc<MPDType>(moment1_out);
  MPDType* moment2_out_data = dev_ctx.template Alloc<MPDType>(moment2_out);

  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision ? dev_ctx.template Alloc<MPDType>(master_param_out)
                      : nullptr;

  MPDType beta1_ = static_cast<MPDType>(beta1);
  MPDType beta2_ = static_cast<MPDType>(beta2);
  MPDType epsilon_ = static_cast<MPDType>(epsilon);

  MPDType rho_inf =
      static_cast<MPDType>(2) / (static_cast<MPDType>(1) - beta2_) -
      static_cast<MPDType>(1);

  // make cpu accumulator to tensor
  DenseTensor beta1_pow_data;
  phi::Copy(dev_ctx, beta1_pow, phi::CPUPlace(), false, &beta1_pow_data);
  MPDType beta1_pow_scalar = beta1_pow_data.data<MPDType>()[0] * beta1_;
  beta1_pow_out_data[0] = beta1_pow_scalar;

  DenseTensor beta2_pow_data;
  phi::Copy(dev_ctx, beta2_pow, phi::CPUPlace(), false, &beta2_pow_data);
  MPDType beta2_pow_scalar = beta2_pow_data.data<MPDType>()[0] * beta2_;
  beta2_pow_out_data[0] = beta2_pow_scalar;

  DenseTensor rho_data;
  phi::Copy(dev_ctx, rho, phi::CPUPlace(), false, &rho_data);
  MPDType rho_scalar =
      (rho_data.data<MPDType>()[0] * (beta2_ - beta2_pow_scalar) +
       beta2_pow_scalar) /
      (static_cast<MPDType>(1) - beta2_pow_scalar);
  rho_out_data[0] = rho_scalar;

  int numel = param.numel();
  int block = 512;
  int grid = (param.numel() + block - 1) / block;
  auto stream = dev_ctx.stream();

  RAdamGPUKernel<T, MPDType>
      <<<block, grid, 0, stream>>>(param.data<T>(),
                                   grad.data<T>(),
                                   learning_rate.data<MPDType>(),
                                   moment1.data<MPDType>(),
                                   moment2.data<MPDType>(),
                                   master_in_data,
                                   beta1_,
                                   beta2_,
                                   epsilon_,
                                   beta1_pow_scalar,
                                   beta2_pow_scalar,
                                   rho_scalar,
                                   rho_inf,
                                   numel,
                                   param_out_data,
                                   moment1_out_data,
                                   moment2_out_data,
                                   master_out_data);
}
}  // namespace phi

PD_REGISTER_KERNEL(radam,
                   GPU,
                   ALL_LAYOUT,
                   phi::RAdamKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
