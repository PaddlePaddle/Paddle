#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/radam_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"

namespace phi {
template <typename T, typename MT>
__global__ void RAdamGPUKernel(const T* param,
                               const T* grad,
                               const MT* learning_rate,
                               const MT* beta1_pow,
                               const MT* beta2_pow,
                               const MT* rho,
                               const MT* moment1,
                               const MT* moment2,
                               const MT* master_param,
                               MT beta1,
                               MT beta2,
                               MT epsilon,
                               MT rho_inf,
                               int num,
                               T* param_out,
                               MT* beta1_pow_out,
                               MT* beta2_pow_out,
                               MT* rho_out,
                               MT* moment1_out,
                               MT* moment2_out,
                               MT* master_param_out) {
  MT lr_scalar = static_cast<MT>(learning_rate[0]);

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int index = idx; index < num; index += gridDim.x * blockDim.x) {
    // load and cast input to MT
    MT d_param =
        master_param ? master_param[index] : static_cast<MT>(param[index]);
    MT d_grad = static_cast<MT>(grad[index]);
    MT d_beta1_pow = static_cast<MT>(beta1_pow[index]);
    MT d_beta2_pow = static_cast<MT>(beta2_pow[index]);
    MT d_rho = static_cast<MT>(rho[index]);
    MT d_moment1 = static_cast<MT>(moment1[index]);
    MT d_moment2 = static_cast<MT>(moment2[index]);

    // compute
    MT beta1_pow_scalar = d_beta1_pow * beta1;
    MT beta2_pow_scalar = d_beta2_pow * beta2;
    MT rho_scalar = (d_rho * (beta2 - beta2_pow_scalar) + beta2_pow_scalar) /
                    (static_cast<MT>(1) - beta2_pow_scalar);

    MT m1_out = beta1 * d_moment1 + (static_cast<MT>(1) - beta1) * d_grad;
    MT m2_out =
        beta2 * d_moment2 + (static_cast<MT>(1) - beta2) * d_grad * d_grad;

    MT m1_hat = m1_out / (static_cast<MT>(1) - beta1_pow_scalar);

    MT rho_t = rho_inf - static_cast<MT>(2) * rho_scalar;
    MT p_out = static_cast<MT>(0);

    if (rho_t > static_cast<MT>(5)) {
      MT l_t = std::sqrt((static_cast<MT>(1) - beta2_pow_scalar)) /
               (std::sqrt(m2_out) + epsilon);
      MT r_t = std::sqrt(((rho_t - static_cast<MT>(4)) *
                          (rho_t - static_cast<MT>(2)) * rho_inf) /
                         ((rho_inf - static_cast<MT>(4)) *
                          (rho_inf - static_cast<MT>(2)) * rho_t));

      p_out = d_param - lr_scalar * m1_hat * r_t * l_t;
    } else {
      p_out = d_param - lr_scalar * m1_hat;
    }

    // store
    param_out[index] = static_cast<T>(p_out);
    beta1_pow_out[index] = static_cast<T>(beta1_pow_scalar);
    beta2_pow_out[index] = static_cast<T>(beta2_pow_scalar);
    rho_out[index] = static_cast<T>(rho_scalar);
    moment1_out[index] = static_cast<MT>(m1_out);
    moment2_out[index] = static_cast<MT>(m2_out);

    if (master_param_out) {
      master_param_out[index] = p_out;
    }
  }
}

template <typename T, typename Context>
void RAdamKernel(const Context& dev_ctx,
                 const DenseTensor& param,
                 const DenseTensor& grad,
                 const DenseTensor& learning_rate,
                 const DenseTensor& beta1_pow,
                 const DenseTensor& beta2_pow,
                 const DenseTensor& rho,
                 const DenseTensor& moment1,
                 const DenseTensor& moment2,
                 const paddle::optional<DenseTensor>& master_param,
                 float beta1,
                 float beta2,
                 float epsilon,
                 bool multi_precision,
                 DenseTensor* param_out,
                 DenseTensor* beta1_pow_out,
                 DenseTensor* beta2_pow_out,
                 DenseTensor* rho_out,
                 DenseTensor* moment1_out,
                 DenseTensor* moment2_out,
                 DenseTensor* master_param_out) {
  using MPDType = typename phi::dtype::template MPTypeTrait<T>::Type;
  T* param_out_data = dev_ctx.template Alloc<T>(param_out);

  MPDType* beta1_pow_out_data = dev_ctx.template Alloc<MPDType>(beta1_pow_out);
  MPDType* beta2_pow_out_data = dev_ctx.template Alloc<MPDType>(beta2_pow_out);
  MPDType* rho_out_data = dev_ctx.template Alloc<MPDType>(rho_out);

  MPDType* moment1_out_data = dev_ctx.template Alloc<MPDType>(moment1_out);
  MPDType* moment2_out_data = dev_ctx.template Alloc<MPDType>(moment2_out);

  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision ? dev_ctx.template Alloc<MPDType>(master_param_out)
                      : nullptr;

  MPDType beta1_ = static_cast<MPDType>(beta1);
  MPDType beta2_ = static_cast<MPDType>(beta2);
  MPDType epsilon_ = static_cast<MPDType>(epsilon);

  MPDType rho_inf =
      static_cast<MPDType>(2) / (static_cast<MPDType>(1) - beta2_) -
      static_cast<MPDType>(1);

  int numel = param.numel();
  int block = 512;
  int grid = (param.numel() + block - 1) / block;
  auto stream = dev_ctx.stream();

  RAdamGPUKernel<T, MPDType>
      <<<block, grid, 0, stream>>>(param.data<T>(),
                                   grad.data<T>(),
                                   learning_rate.data<MPDType>(),
                                   beta1_pow.data<MPDType>(),
                                   beta2_pow.data<MPDType>(),
                                   rho.data<MPDType>(),
                                   moment1.data<MPDType>(),
                                   moment2.data<MPDType>(),
                                   master_in_data,
                                   beta1_,
                                   beta2_,
                                   epsilon_,
                                   rho_inf,
                                   numel,
                                   param_out_data,
                                   beta1_pow_out_data,
                                   beta2_pow_out_data,
                                   rho_out_data,
                                   moment1_out_data,
                                   moment2_out_data,
                                   master_out_data);
}
}  // namespace phi

PD_REGISTER_KERNEL(radam,
                   GPU,
                   ALL_LAYOUT,
                   phi::RAdamKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
