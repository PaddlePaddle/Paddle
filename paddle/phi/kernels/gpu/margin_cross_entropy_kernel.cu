#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// old op include, fluid should be removed
#ifdef PADDLE_WITH_HIP
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#else
#include <hipcub/hipcub.hpp>
#endif

#include <vector>
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
#include "paddle/fluid/distributed/collective/ProcessGroup.h"
#include "paddle/fluid/platform/collective_helper.h"
#include "paddle/fluid/platform/device/gpu/nccl_helper.h"
#endif
// trace op include
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename Context>
void GetClassInterval(const gpuStream_t& stream,
                      const phi::Place& place,
                      const Context& dev_ctx,
                      const int rid,
                      const int rank,
                      const int nranks,
                      const int D,
                      DenseTensor* class_interval) {
  std::vector<int> shard_dim_vec(nranks + 1, 0);
  shard_dim_vec[rank + 1] = D;
  if (nranks <= 1) {
    paddle::framework::TensorFromVector(shard_dim_vec, dev_ctx, class_interval);
    return;
  }

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  DenseTensor num_classes_per_device;
  paddle::framework::TensorFromVector(
      shard_dim_vec, dev_ctx, &num_classes_per_device);
  int* num_classes_per_device_ptr = num_classes_per_device.data<int>();

  auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();
  if (map->has(rid)) {
    // Use ProcessGroup
    paddle::distributed::ProcessGroup* pg = map->get(rid);
    std::vector<phi::DenseTensor> in_tensor;
    std::vector<phi::DenseTensor> out_tensor;
    in_tensor.push_back(num_classes_per_device);
    out_tensor.push_back(num_classes_per_device);

    paddle::distributed::AllreduceOptions opts;
    opts.reduce_op = paddle::distributed::ReduceOp::SUM;
    auto task = pg->AllReduce(in_tensor, out_tensor, opts);
    task->Wait();
  } else {
    const auto& comm =
        paddle::platform::NCCLCommContext::Instance().Get(rid, place);
    // use global calculate stream
    const auto calcu_stream =
        static_cast<GPUContext*>(
            paddle::platform::DeviceContextPool::Instance().Get(place))
            ->stream();

    PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::ncclAllReduce(
        num_classes_per_device_ptr,
        num_classes_per_device_ptr,
        num_classes_per_device.numel(),
        paddle::platform::ToNCCLDataType(paddle::framework::TransToProtoVarType(
            num_classes_per_device.dtype())),
        ncclSum,
        comm->comm(),
        calcu_stream));
  }

  class_interval->Resize({nranks + 1});
  auto class_interval_ptr = dev_ctx.template Alloc<int>(class_interval);
  size_t cub_temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum<int*, int*>(
      nullptr, cub_temp_storage_bytes, nullptr, nullptr, nranks + 1, stream);
  auto cub_temp_storage = paddle::memory::Alloc(place, cub_temp_storage_bytes);
  hipcub::DeviceScan::InclusiveSum<int*, int*>(cub_temp_storage->ptr(),
                                            cub_temp_storage_bytes,
                                            num_classes_per_device_ptr,
                                            class_interval_ptr,
                                            nranks + 1,
                                            stream);
  return;
#endif
}

template <typename T, typename IndexT>
__global__ void AddMarginToPositiveLogitsKernel(T* logit,
                                                const IndexT* label,
                                                const float margin1,
                                                const float margin2,
                                                const float margin3,
                                                const int rank,
                                                const int nranks,
                                                const int64_t N,
                                                const int64_t D,
                                                const int* class_interval_ptr) {
  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  int start_index = class_interval_ptr[rank];
  int end_index = class_interval_ptr[rank + 1];
  int num_classes = class_interval_ptr[nranks];
  CUDA_KERNEL_LOOP(i, N) {
    auto real_label = label[i];
    PADDLE_ENFORCE((real_label < num_classes) && (real_label >= 0),
                   "The index is out of bounds, "
                   "please check whether the value of label and "
                   "input meet the number of class. It should "
                   "be less than [%d], but received [%d]",
                   num_classes,
                   real_label);

    if (real_label >= start_index && real_label < end_index) {
      int64_t offset = i * D + real_label - start_index;
      if (fabs(margin1 - 1.0) > 1e-8 || fabs(margin2) > 1e-8) {
        MPType x = static_cast<MPType>(logit[offset]);
        MPType theta = acos(x);
        if (fabs(margin1 - 1.0) > 1e-8) {
          theta *= static_cast<MPType>(margin1);
        }
        if (fabs(margin2) > 1e-8) {
          theta += static_cast<MPType>(margin2);
        }
        logit[offset] = static_cast<T>(cos(theta));
      }
      if (fabs(margin3) > 1e-8) {
        MPType y = static_cast<MPType>(logit[offset]);
        y -= static_cast<MPType>(margin3);
        logit[offset] = static_cast<T>(y);
      }
    }
  }
}

template <typename T>
__global__ void ScaleLogitKernel(T* logits,
                                 const float scale,
                                 const int64_t N,
                                 const int64_t D) {
  CUDA_KERNEL_LOOP(i, N * D) { logits[i] *= static_cast<T>(scale); }
}

template <typename T>
__global__ void LogitsMinusMaxKernel(T* logits,
                                     const T* logits_max_per_row,
                                     const int64_t N,
                                     const int64_t D) {
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    logits[i] -= logits_max_per_row[row];
  }
}

template <typename T>
__global__ void LogitsMinusLogSumKernel(T* logits,
                                        const T* logits_sum_per_row,
                                        const int64_t N,
                                        const int64_t D) {
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    logits[i] -= phi::kps::details::Log(logits_sum_per_row[row]);
  }
}

template <typename T, typename IndexT>
__global__ void HardLabelSoftmaxWithCrossEntropyKernel(
    T* loss,
    T* log_softmax,
    const IndexT* labels,
    const int rank,
    const int64_t N,
    const int64_t D,
    const int* class_interval_ptr) {
  int start_index = class_interval_ptr[rank];
  CUDA_KERNEL_LOOP(i, N * D) {
    auto row = i / D;
    auto col = i % D;
    if ((col + start_index) == labels[row]) {
      auto softmax = log_softmax[i];
      loss[row] = -softmax;
      log_softmax[i] = phi::kps::details::Exp(softmax);
    } else {
      log_softmax[i] = phi::kps::details::Exp(log_softmax[i]);
    }
  }
}

template <typename T, typename Context>
void MarginCrossEntropyKernel(const Context& dev_ctx,
                              const DenseTensor& logits,
                              const DenseTensor& labels,
                              bool return_softmax,
                              int ring_id,
                              int rank,
                              int nranks,
                              float margin1,
                              float margin2,
                              float margin3,
                              float scale,
                              DenseTensor* softmax,
                              DenseTensor* loss) {
  const auto& place = dev_ctx.GetPlace();  // old code

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  paddle::platform::NCCLComm* comm;
  paddle::distributed::ProcessGroup* pg = nullptr;
  gpuStream_t stream;
  if (nranks > 1) {
    auto map = paddle::distributed::ProcessGroupMapFromGid::getInstance();
    if (map->has(ring_id)) {
      // Use ProcessGroup
      pg = map->get(ring_id);
    } else {
      comm = paddle::platform::NCCLCommContext::Instance().Get(ring_id, place);

      // use global calculate stream
      stream = static_cast<GPUContext*>(
                   paddle::platform::DeviceContextPool::Instance().Get(place))
                   ->stream();
    }
  }
#endif

  // allocate memory on device.
  T* softmax_ptr = dev_ctx.template Alloc<T>(softmax);
  T* loss_ptr = dev_ctx.template Alloc<T>(loss);

  const auto& logits_dims = logits.dims();
  const auto& labels_dims = labels.dims();

  const int axis = logits_dims.size() - 1;
  const int N = phi::funcs::SizeToAxis(axis, logits_dims);
  const int D = phi::funcs::SizeFromAxis(axis, logits_dims);

  int blocks = NumBlocks(N);
  int threads = kNumCUDAThreads;
  const auto& label_type =
      paddle::framework::TransToProtoVarType(labels.dtype());

  // copy logits to softmax variable since we can't modify logits,
  // and it also be used when calculate grad
  phi::Copy<Context>(dev_ctx, logits, dev_ctx.GetPlace(), true, softmax);

  DenseTensor softmax_2d;
  softmax_2d.ShareDataWith(*softmax).Resize({N, D});
  T* logits_ptr = softmax_2d.data<T>();

  DenseTensor class_interval;
  GetClassInterval<T, Context>(dev_ctx.stream(),
                               dev_ctx.GetPlace(),
                               dev_ctx,
                               ring_id,
                               rank,
                               nranks,
                               D,
                               &class_interval);

  // step 1, preprocess logits
  // add margin for positive elements
  // theta = acos(x_i)
  // (cos(m1 * theta + m2) - m3)
  // save match_logits, used for gradient computation.
  if (label_type == paddle::framework::proto::VarType::INT32) {
    typedef int32_t LabelT;
    AddMarginToPositiveLogitsKernel<T>
        <<<NumBlocks(N), threads, 0, dev_ctx.stream()>>>(
            logits_ptr,
            labels.data<LabelT>(),
            margin1,
            margin2,
            margin3,
            rank,
            nranks,
            N,
            D,
            class_interval.data<int>());
  } else if (label_type == paddle::framework::proto::VarType::INT64) {
    typedef int64_t LabelT;
    AddMarginToPositiveLogitsKernel<T>
        <<<NumBlocks(N), threads, 0, dev_ctx.stream()>>>(
            logits_ptr,
            labels.data<LabelT>(),
            margin1,
            margin2,
            margin3,
            rank,
            nranks,
            N,
            D,
            class_interval.data<int>());
  } else {
    PADDLE_THROW(errors::Unimplemented(
        "margin_cross_entropy label type noly support int32 and int64, "
        "but got %s",
        label_type));
  }

  // scale by s
  ScaleLogitKernel<T><<<NumBlocks(N * D), threads, 0, dev_ctx.stream()>>>(
      logits_ptr, scale, N, D);

  // step 2, obtain logit_max
  DenseTensor logits_max;
  logits_max.Resize({N, 1});
  dev_ctx.template Alloc<T>(&logits_max);
  T* logits_max_buff = dev_ctx.template Alloc<T>(&logits_max);

  phi::funcs::
      ReduceKernel<T, T, phi::kps::MaxFunctor, phi::kps::IdentityFunctor<T>>(
          static_cast<const phi::GPUContext&>(dev_ctx),
          softmax_2d,
          &logits_max,
          phi::kps::IdentityFunctor<T>(),
          {1});

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    if (pg) {
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(logits_max);
      out_tensor.push_back(logits_max);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::MAX;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::ncclAllReduce(
          logits_max_buff,
          logits_max_buff,
          logits_max.numel(),
          paddle::platform::ToNCCLDataType(
              paddle::framework::TransToProtoVarType(logits_max.dtype())),
          ncclMax,
          comm->comm(),
          stream));
    }
  }
#endif

  // step 3, logit - logit_max
  LogitsMinusMaxKernel<T><<<NumBlocks(N * D), threads, 0, dev_ctx.stream()>>>(
      logits_ptr, logits_max_buff, N, D);

  // step 4, sum(exp(logit - logit_max))
  DenseTensor sum_exp_logits;
  sum_exp_logits.Resize({N, 1});
  dev_ctx.template Alloc<T>(&sum_exp_logits);
  T* sum_exp_logits_buff = dev_ctx.template Alloc<T>(&sum_exp_logits);
  phi::funcs::ReduceKernel<T, T, phi::kps::AddFunctor, phi::kps::ExpFunctor<T>>(
      static_cast<const phi::GPUContext&>(dev_ctx),
      softmax_2d,
      &sum_exp_logits,
      phi::kps::ExpFunctor<T>(),
      {1});

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    if (pg) {
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(sum_exp_logits);
      out_tensor.push_back(sum_exp_logits);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::SUM;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::ncclAllReduce(
          sum_exp_logits_buff,
          sum_exp_logits_buff,
          sum_exp_logits.numel(),
          paddle::platform::ToNCCLDataType(
              paddle::framework::TransToProtoVarType(sum_exp_logits.dtype())),
          ncclSum,
          comm->comm(),
          stream));
    }
  }
#endif

  // step 5, (logit - logit_max) - log(sum(exp(logit - logit_max)))
  LogitsMinusLogSumKernel<T>
      <<<NumBlocks(N * D), threads, 0, dev_ctx.stream()>>>(
          logits_ptr, sum_exp_logits_buff, N, D);

  // step 6, prob = exp((logit - logit_max) - log(sum(exp(logit -
  // logit_max))))
  // loss = -((logit_i - logit_max) - log(sum(exp(logit - logit_max))))

  phi::funcs::SetConstant<Context, T> functor;
  functor(dev_ctx, loss, static_cast<T>(0.0));
  if (label_type == paddle::framework::proto::VarType::INT32) {
    typedef int32_t LabelT;
    HardLabelSoftmaxWithCrossEntropyKernel<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_ptr,
                                                   logits_ptr,
                                                   labels.data<LabelT>(),
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  } else if (label_type == paddle::framework::proto::VarType::INT64) {
    typedef int64_t LabelT;
    HardLabelSoftmaxWithCrossEntropyKernel<T, LabelT>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(loss_ptr,
                                                   logits_ptr,
                                                   labels.data<LabelT>(),
                                                   rank,
                                                   N,
                                                   D,
                                                   class_interval.data<int>());
  }

#if defined(PADDLE_WITH_NCCL) || defined(PADDLE_WITH_RCCL)
  if (nranks > 1) {
    if (pg) {
      std::vector<phi::DenseTensor> in_tensor;
      std::vector<phi::DenseTensor> out_tensor;
      in_tensor.push_back(*loss);
      out_tensor.push_back(*loss);

      paddle::distributed::AllreduceOptions opts;
      opts.reduce_op = paddle::distributed::ReduceOp::SUM;
      auto task = pg->AllReduce(in_tensor, out_tensor, opts);
      task->Wait();
    } else {
      PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::ncclAllReduce(
          loss_ptr,
          loss_ptr,
          loss->numel(),
          paddle::platform::ToNCCLDataType(
              paddle::framework::TransToProtoVarType(loss->dtype())),
          ncclSum,
          comm->comm(),
          stream));
    }
  }
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(margin_cross_entropy,
                   GPU,
                   ALL_LAYOUT,
                   phi::MarginCrossEntropyKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
