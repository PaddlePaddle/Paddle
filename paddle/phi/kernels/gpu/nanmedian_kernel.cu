#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/nanmedian_kernel.h"

#include "paddle/fluid/memory/memcpy.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/full_kernel.h"
#include "paddle/phi/kernels/impl/nanmedian_kernel_impl.h"
#include "paddle/phi/kernels/top_k_kernel.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T>
__global__ void KernelNanCounts(const T* input,
                                const int numel,
                                const int64_t pre_dim,
                                const int64_t stride,
                                T min_val,
                                int64_t* nan_total,
                                int64_t* nan_counts) {
  extern __shared__ int64_t buf[];
  for (int i = threadIdx.x; i < pre_dim; i += blockDim.x) {
    buf[i] = 0;
    nan_counts[i] = 0;
  }

  if (threadIdx.x == 0) {
    nan_total[0] = 0;
    nan_total[1] = 0;
  }

  __syncthreads();

  CUDA_KERNEL_LOOP(index, numel) {
    const T x = input[index];
    if (isnan(static_cast<float>(x))) {
      auto bin = static_cast<int64_t>(index / stride);
      paddle::platform::CudaAtomicAdd(&buf[bin], 1);
    }
  }
  __syncthreads();

  for (int i = threadIdx.x; i < pre_dim; i += blockDim.x) {
    paddle::platform::CudaAtomicAdd(&nan_counts[i], buf[i]);
    paddle::platform::CudaAtomicAdd(&nan_total[0], buf[i]);
    paddle::platform::CudaAtomicMax(&nan_total[1], stride - buf[i]);
  }
}

template <typename T>
__global__ void CalcMedianKernel(const T* sort_out_ptr,
                                 const int64_t* sort_indices_ptr,
                                 int64_t* median_val,
                                 T* output,
                                 T div_factor,
                                 const bool is_odd,
                                 const int64_t pre_dim,
                                 const int64_t stride) {
  CUDA_KERNEL_LOOP(index, pre_dim) {
    int64_t pos = static_cast<int64_t>((index + 1) * stride) - 1;
    if (is_odd) {
      median_val[index * 2] = sort_indices_ptr[pos];
      median_val[index * 2 + 1] = sort_indices_ptr[pos];
      output[index] = sort_out_ptr[pos];
    } else {
      median_val[index * 2] =
          pos > 0 ? sort_indices_ptr[pos - 1] : sort_indices_ptr[pos];
      median_val[index * 2 + 1] = sort_indices_ptr[pos];
      T median_val_left = pos > 0 ? sort_out_ptr[pos - 1] : sort_out_ptr[pos];
      T median_val_right = sort_out_ptr[pos];
      output[index] = (median_val_left + median_val_right) / div_factor;
    }
  }
}

template <typename T>
__global__ void CalcNanmedianKernel(const T* sort_out_ptr,
                                    const int64_t* sort_indices_ptr,
                                    int64_t* nan_counts,
                                    int64_t* median_val,
                                    T* output,
                                    const bool is_odd,
                                    const int64_t pre_dim,
                                    const int64_t max_valid_num,
                                    const int64_t stride,
                                    const T div_factor,
                                    const T nan_val) {
  CUDA_KERNEL_LOOP(index, pre_dim) {
    int64_t pos = static_cast<int64_t>(index * max_valid_num);
    int64_t nan_cnt = nan_counts[index];
    if (nan_cnt == stride) {
      median_val[index * 2] = -1;
      median_val[index * 2 + 1] = -1;
      output[index] = nan_val;
    } else {
      int64_t nan_k =
          nan_cnt > 0 ? static_cast<int64_t>(stride - nan_cnt) : max_valid_num;
      int64_t row_pos = static_cast<int64_t>(nan_k >> 1);
      pos += row_pos;

      if (nan_k & 1) {
        median_val[index * 2] = sort_indices_ptr[pos];
        median_val[index * 2 + 1] = sort_indices_ptr[pos];
        output[index] = sort_out_ptr[pos];
      } else {
        median_val[index * 2] =
            pos > 0 ? sort_indices_ptr[pos - 1] : sort_indices_ptr[pos];
        median_val[index * 2 + 1] = sort_indices_ptr[pos];
        T median_val_left = pos > 0 ? sort_out_ptr[pos - 1] : sort_out_ptr[pos];
        T median_val_right = sort_out_ptr[pos];
        output[index] = (median_val_left + median_val_right) / div_factor;
      }
    }
  }
}

template <typename T, typename Context>
void ProcessMedianKernel(const Context& dev_ctx,
                         const DenseTensor& x,
                         bool ignore_nan,
                         DenseTensor* out,
                         int64_t* m_ptr) {
  bool should_ignore_nan = ignore_nan;
  auto stream = dev_ctx.stream();

  const T* x_ptr = x.data<T>();
  T* o_ptr = dev_ctx.template Alloc<T>(out);
  int64_t numel = x.numel();
  auto x_dim = x.dims();
  int64_t x_rank = x_dim.size();
  int64_t stride = x_dim[x_rank - 1];
  int64_t pre_dim = numel / stride;
  int64_t i = 0;

  DenseTensor nan_counts, nan_stat;
  int64_t* nan_counts_ptr;
  int64_t max_valid_num = 0;
  if (should_ignore_nan) {
    nan_counts.Resize(phi::make_ddim({pre_dim}));
    dev_ctx.template Alloc<int64_t>(&nan_counts);
    nan_counts_ptr = nan_counts.data<int64_t>();
    nan_stat.Resize(phi::make_ddim({2}));
    int64_t* nan_stat_mem = dev_ctx.template Alloc<int64_t>(&nan_stat);
    int64_t* nan_stat_ptr = nan_stat.data<int64_t>();

    KernelNanCounts<T><<<GET_BLOCKS(numel),
                         PADDLE_CUDA_NUM_THREADS,
                         pre_dim * sizeof(int64_t),
                         stream>>>(x_ptr,
                                   numel,
                                   pre_dim,
                                   stride,
                                   std::numeric_limits<T>::min(),
                                   nan_stat_ptr,
                                   nan_counts_ptr);

    auto nan_stat_mem_cpu =
        paddle::memory::Alloc(phi::CPUPlace(), sizeof(int64_t) * 2);
    int64_t* nan_stat_cpu_ptr =
        reinterpret_cast<int64_t*>(nan_stat_mem_cpu->ptr());
    paddle::memory::Copy(phi::CPUPlace(),
                         nan_stat_cpu_ptr,
                         dev_ctx.GetPlace(),
                         nan_stat_mem,
                         sizeof(int64_t) * 2,
                         stream);

    // all elements are nan values
    T nan_val = std::numeric_limits<T>::quiet_NaN();
    if (nan_stat_cpu_ptr[0] == numel) {
      FullLikeKernel<T, Context>(dev_ctx, x, nan_val, x.dtype(), out);
      return;
    }

    should_ignore_nan = nan_stat_cpu_ptr[0] > 0;
    max_valid_num = nan_stat_cpu_ptr[1];
  }

  int64_t sort_k = should_ignore_nan ? max_valid_num : ((stride >> 1) + 1);
  bool is_ori_odd = stride & 1;

  DenseTensor sort_out, sort_indices;
  auto sort_dim = x.dims();
  int64_t rank = sort_dim.size();
  sort_dim[rank - 1] = sort_k;
  sort_out.Resize(sort_dim);
  sort_indices.Resize(sort_dim);

  dev_ctx.template Alloc<T>(&sort_out);
  T* sort_out_ptr = sort_out.data<T>();
  dev_ctx.template Alloc<int64_t>(&sort_indices);
  int64_t* sort_indices_ptr = sort_indices.data<int64_t>();

  TopkKernel<T, Context>(
      dev_ctx, x, Scalar(sort_k), -1, false, true, &sort_out, &sort_indices);

  T div_factor = static_cast<T>(2.0);
  T nan_val = std::numeric_limits<T>::quiet_NaN();
  if (should_ignore_nan) {
    CalcNanmedianKernel<T>
        <<<GET_BLOCKS(pre_dim), PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
            sort_out_ptr,
            sort_indices_ptr,
            nan_counts_ptr,
            m_ptr,
            o_ptr,
            is_ori_odd,
            pre_dim,
            max_valid_num,
            stride,
            div_factor,
            nan_val);
  } else {
    CalcMedianKernel<T>
        <<<GET_BLOCKS(pre_dim), PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
            sort_out_ptr,
            sort_indices_ptr,
            m_ptr,
            o_ptr,
            div_factor,
            is_ori_odd,
            pre_dim,
            sort_k);
  }
}

template <typename T, typename Context>
void BaseMedianKernel(const Context& dev_ctx,
                      const DenseTensor& input,
                      const IntArray& axes,
                      bool ignore_nan,
                      DenseTensor* out,
                      DenseTensor* median_index) {
  DenseTensor x;
  auto rank = input.dims().size();
  if ((axes.size() == 0) || rank <= 1) {
    x = input;
    x.Resize({input.numel()});
  } else {
    PreprocessMedianKernel<T, Context>(dev_ctx, input, axes, &x);
  }

  int64_t* m_ptr = dev_ctx.template Alloc<int64_t>(median_index);
  ProcessMedianKernel<T, Context>(dev_ctx, x, ignore_nan, out, m_ptr);
  out->Resize(out->dims());
}

template <typename T, typename Context>
void NanmedianKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const IntArray& axes,
                     bool keepdim,
                     DenseTensor* out,
                     DenseTensor* median_index) {
  BaseMedianKernel<T, Context>(dev_ctx, x, axes, true, out, median_index);
}

}  // namespace phi

PD_REGISTER_KERNEL(nanmedian,
                   GPU,
                   ALL_LAYOUT,
                   phi::NanmedianKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
