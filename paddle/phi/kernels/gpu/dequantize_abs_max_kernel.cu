#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/dequantize_abs_max_kernel.h"

#include "paddle/common/hostdevice.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math.h"

namespace phi {

template <typename T>
__global__ void KeDequantize(
    const T* in, const float* scale, float max_range, int num, float* out) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < num) {
    out[idx] = in[idx] * scale[0] / max_range;
  }
}

template <typename T, typename Context>
void DequantizeAbsMaxKernel(const Context& dev_ctx,
                            const DenseTensor& x,
                            const DenseTensor& scale,
                            float max_range,
                            DenseTensor* out) {
  const T* in_data = x.data<T>();
  const float* scale_factor = scale.data<float>();
  float* out_data = dev_ctx.template Alloc<float>(out);

  int num = x.numel();
  int block = 512;
  int grid = (num + block - 1) / block;

  KeDequantize<T><<<grid, block, 0, dev_ctx.stream()>>>(
      in_data, scale_factor, max_range, num, out_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(dequantize_abs_max,
                   GPU,
                   ALL_LAYOUT,
                   ops::DequantizeAbsMaxKernel,
                   int8_t,
                   int16_t) {}
