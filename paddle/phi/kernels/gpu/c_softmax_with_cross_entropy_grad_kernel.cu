#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/cross_entropy.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/math.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/softmax.h"
#include "paddle/phi/kernels/funcs/softmax_impl.h"
#include "paddle/phi/kernels/reduce_sum_kernel.h"
#include "paddle/utils/string/string_helper.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int64_t kNumMaxinumNumBlocks = 4096;

static inline int64_t NumBlocks(const int64_t N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T, typename IndexT>
__global__ void MaskLabelByIndexGrad(T* logits_grad,
                                     const T* loss_grad,
                                     const IndexT* labels,
                                     const int64_t start_index,
                                     const int64_t end_index,
                                     const int64_t N,
                                     const int64_t D,
                                     const int64_t ignore_index) {
  CUDA_KERNEL_LOOP_TYPE(i, N * D, int64_t) {
    auto row = i / D;
    auto col = i % D;
    auto lbl = static_cast<int64_t>(labels[row]);
    if (lbl == ignore_index) {
      logits_grad[i] = static_cast<T>(0.0);
    } else if ((col + start_index) == labels[row]) {
      logits_grad[i] = (logits_grad[i] - static_cast<T>(1.0)) * loss_grad[row];
    } else {
      logits_grad[i] *= loss_grad[row];
    }
  }
}

template <typename T, typename Context>
void CSoftmaxWithCrossEntropyGradKernel(const Context& dev_ctx,
                                        const DenseTensor& softmax_in,
                                        const DenseTensor& label_in,
                                        const DenseTensor& loss_grad_in,
                                        int64_t ignore_index,
                                        int ring_id,
                                        int rank,
                                        int nranks,
                                        DenseTensor* logits_grad) {
  const phi::DenseTensor* labels = &label_in;
  const phi::DenseTensor* loss_grad = &loss_grad_in;
  const phi::DenseTensor* softmax = &softmax_in;
  phi::DenseTensor* logit_grad = logits_grad;

  if (logit_grad != softmax) {
    phi::Copy(dev_ctx, *softmax, dev_ctx.GetPlace(), false, logit_grad);
  }
  const auto sofrmax_dims = softmax->dims();
  const int axis = sofrmax_dims.size() - 1;
  const int64_t N = phi::funcs::SizeToAxis<int64_t>(axis, sofrmax_dims);
  const int64_t D = phi::funcs::SizeFromAxis<int64_t>(axis, sofrmax_dims);

  phi::DenseTensor logit_grad_2d;
  logit_grad_2d.ShareDataWith(*logit_grad).Resize({N, D});

  int64_t blocks = NumBlocks(N * D);
  int threads = kNumCUDAThreads;
  const auto& label_type = labels->dtype();
  const int64_t start_index = rank * D;
  const int64_t end_index = start_index + D;

  if (label_type == phi::DataType::INT32) {
    MaskLabelByIndexGrad<T, int32_t>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logit_grad_2d.data<T>(),
                                                   loss_grad->data<T>(),
                                                   labels->data<int32_t>(),
                                                   start_index,
                                                   end_index,
                                                   N,
                                                   D,
                                                   ignore_index);
  } else if (label_type == phi::DataType::INT64) {
    MaskLabelByIndexGrad<T, int64_t>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(logit_grad_2d.data<T>(),
                                                   loss_grad->data<T>(),
                                                   labels->data<int64_t>(),
                                                   start_index,
                                                   end_index,
                                                   N,
                                                   D,
                                                   ignore_index);
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(c_softmax_with_cross_entropy_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CSoftmaxWithCrossEntropyGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
