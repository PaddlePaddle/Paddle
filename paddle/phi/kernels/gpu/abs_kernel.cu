#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <vector>
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/abs_kernel.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"

namespace phi {

template <typename T, typename Enable = void>
struct CudaAbsFunctor;

template <typename T>
struct CudaAbsFunctor<T, phi::funcs::Complex<T, phi::dtype::Real<T>>> {
  __device__ __forceinline__ phi::dtype::Real<T> operator()(const T x) const {
    return abs(x);
  }
};

template <typename T>
struct CudaAbsFunctor<T, phi::funcs::NoComplex<T, phi::dtype::Real<T>>> {
  __device__ __forceinline__ T operator()(const T x) const {
    return std::abs(x);
  }
};

template <typename T, typename Context>
void AbsKernel(const Context& ctx, const DenseTensor& x, DenseTensor* out) {
  ctx.template Alloc<phi::dtype::Real<T>>(out);
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  auto functor = CudaAbsFunctor<T>();

  funcs::ElementwiseKernel<phi::dtype::Real<T>>(ctx, ins, &outs, functor);
}

}  // namespace phi

PD_REGISTER_KERNEL(abs,
                   GPU,
                   ALL_LAYOUT,
                   phi::AbsKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
