#include "hip/hip_runtime.h"
// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/anchor_generator_kernel_impl.h"

namespace phi {

template <typename T>
__global__ void GenAnchors(T* out,
                           const T* aspect_ratios,
                           const int ar_num,
                           const T* anchor_sizes,
                           const int as_num,
                           const T* stride,
                           const int sd_num,
                           const int height,
                           const int width,
                           const T offset) {
  int num_anchors = as_num * ar_num;
  int box_num = height * width * num_anchors;
  CUDA_KERNEL_LOOP(i, box_num) {
    int h_idx = i / (num_anchors * width);
    int w_idx = (i / num_anchors) % width;
    T stride_width = stride[0];
    T stride_height = stride[1];
    T x_ctr = (w_idx * stride_width) + offset * (stride_width - 1);
    T y_ctr = (h_idx * stride_height) + offset * (stride_height - 1);
    T area, area_ratios;
    T base_w, base_h;
    T scale_w, scale_h;
    T anchor_width, anchor_height;
    int anch_idx = i % num_anchors;
    int ar_idx = anch_idx / as_num;
    int as_idx = anch_idx % as_num;
    T aspect_ratio = aspect_ratios[ar_idx];
    T anchor_size = anchor_sizes[as_idx];
    area = stride_width * stride_height;
    area_ratios = area / aspect_ratio;
    base_w = round(sqrt(area_ratios));
    base_h = round(base_w * aspect_ratio);
    scale_w = anchor_size / stride_width;
    scale_h = anchor_size / stride_height;
    anchor_width = scale_w * base_w;
    anchor_height = scale_h * base_h;

    T xmin = (x_ctr - .5f * (anchor_width - 1));
    T ymin = (y_ctr - .5f * (anchor_height - 1));
    T xmax = (x_ctr + .5f * (anchor_width - 1));
    T ymax = (y_ctr + .5f * (anchor_height - 1));
    reinterpret_cast<float4*>(out)[i] = make_float4(xmin, ymin, xmax, ymax);
  }
}

template <typename T>
__global__ void SetVariance(T* out,
                            const T* var,
                            const int vnum,
                            const int num) {
  CUDA_KERNEL_LOOP(i, num) { out[i] = var[i % vnum]; }
}

template <typename T, typename Context>
void AnchorGeneratorOpCUDAKernel(const Context& dev_ctx,
                                 const DenseTensor& input_in,
                                 const std::vector<float>& anchor_sizes,
                                 const std::vector<float>& aspect_ratios,
                                 const std::vector<float>& variances,
                                 const std::vector<float>& stride,
                                 float offset_in,
                                 DenseTensor* anchors_out,
                                 DenseTensor* variances_out) {
  auto* input = &input_in;
  auto* anchors = anchors_out;
  auto* vars = variances_out;

  T offset = static_cast<T>(offset_in);

  auto width = input->dims()[3];
  auto height = input->dims()[2];

  int num_anchors = aspect_ratios.size() * anchor_sizes.size();

  int box_num = width * height * num_anchors;

  int block = 512;
  int grid = (box_num + block - 1) / block;

  auto stream = dev_ctx.stream();

  dev_ctx.template Alloc<T>(anchors);
  dev_ctx.template Alloc<T>(vars);

  phi::DenseTensor ar;
  framework::TensorFromVector(aspect_ratios, dev_ctx, &ar);

  phi::DenseTensor as;
  framework::TensorFromVector(anchor_sizes, dev_ctx, &as);

  phi::DenseTensor sd;
  framework::TensorFromVector(stride, dev_ctx, &sd);

  GenAnchors<T><<<grid, block, 0, stream>>>(anchors->data<T>(),
                                            ar.data<T>(),
                                            aspect_ratios.size(),
                                            as.data<T>(),
                                            anchor_sizes.size(),
                                            sd.data<T>(),
                                            stride.size(),
                                            height,
                                            width,
                                            offset);

  phi::DenseTensor v;
  framework::TensorFromVector(variances, dev_ctx, &v);
  grid = (box_num * 4 + block - 1) / block;
  SetVariance<T><<<grid, block, 0, stream>>>(
      vars->data<T>(), v.data<T>(), variances.size(), box_num * 4);
}

}  // namespace phi

namespace ops = paddle::operators;

PD_REGISTER_KERNEL(anchor_generator,
                   GPU,
                   ALL_LAYOUT,
                   phi::AnchorGeneratorOpCUDAKernel,
                   float,
                   double) {}
