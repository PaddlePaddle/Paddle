#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/mean_all_kernel.h"

namespace phi {

template <typename T>
__global__ void MeanRunKernel(const T* in_data, T* out_data, int N) {
  using MT = typename dtype::MPTypeTrait<T>::Type;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  auto data = static_cast<MT>(in_data[0]);
  for (; idx < N; idx += blockDim.x * gridDim.x) {
    out_data[idx] = static_cast<T>(data / (static_cast<MT>(N)));
  }
}

template <typename T, typename Context>
void MeanAllGradKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const DenseTensor& out_grad,
                       DenseTensor* x_grad) {
  PADDLE_ENFORCE_EQ(out_grad.numel(),
                    1,
                    phi::errors::InvalidArgument(
                        "Mean Gradient Input Tensor len should be 1. But "
                        "received Out@Grad's elements num is %d.",
                        out_grad.numel()));
  dev_ctx.template Alloc<T>(x_grad);

  auto in_data = out_grad.data<T>();
  auto size_prob = x_grad->numel();
  auto out_data = x_grad->data<T>();
  int threads = 512;
  int grid = (size_prob + threads - 1) / threads;
  auto stream = dev_ctx.stream();
  MeanRunKernel<T><<<grid, threads, 0, stream>>>(in_data, out_data, size_prob);
}

}  // namespace phi

PD_REGISTER_KERNEL(mean_all_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::MeanAllGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
