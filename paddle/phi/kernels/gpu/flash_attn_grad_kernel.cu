// 2024 - Modified by MetaX Integrated Circuits (Shanghai) Co., Ltd. All Rights Reserved.   
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_grad_kernel.h"
#include "glog/logging.h"  // For VLOG()
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/gpu/flash_attn_utils.h"

namespace phi {

template <typename T, typename Context>
void FlashAttnUnpaddedGradKernel(const Context& ctx,
                                 const DenseTensor& q,
                                 const DenseTensor& k,
                                 const DenseTensor& v,
                                 const DenseTensor& cu_seqlens_q,
                                 const DenseTensor& cu_seqlens_k,
                                 const DenseTensor& out,
                                 const DenseTensor& softmax_lse,
                                 const DenseTensor& seed_offset,
                                 const paddle::optional<DenseTensor>& attn_mask,
                                 const DenseTensor& dout,
                                 int64_t max_seqlen_q,
                                 int64_t max_seqlen_k,
                                 float scale,
                                 float dropout,
                                 bool causal,
                                 DenseTensor* dq,
                                 DenseTensor* dk,
                                 DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(dq);
  DenseTensor dk_tmp;
  if (dk) {
    ctx.template Alloc<T>(dk);
    dk_tmp = *dk;
  } else {
    dk_tmp = EmptyLike<T, Context>(ctx, k);
  }

  DenseTensor dv_tmp;
  if (dv) {
    ctx.template Alloc<T>(dv);
    dv_tmp = *dv;
  } else {
    dv_tmp = EmptyLike<T, Context>(ctx, v);
  }

  const hipStream_t stream = ctx.stream();

  // q,k,v [total_*, num_heads, head_dim]
  auto dims = q.dims();

  const int64_t batch_size = cu_seqlens_q.numel() - 1;
  const int64_t num_heads = dims[1];
  const int64_t head_size_og = dout.dims()[2];
  const int64_t head_size = dims[2];
  const int64_t num_heads_k = k.dims()[1];
  const int64_t total_q = dims[0];
  const int64_t total_k = k.dims()[0];

  // TODO(umiswing): add shape check
  PADDLE_ENFORCE_EQ(
      head_size_og,
      head_size,
      phi::errors::InvalidArgument(
          "flash_attn_bwd receive input with head_size_og == head_size"));

  FlashAttnParamsBwd params = FlashAttnParamsBwd(ctx,
                                                 attn_mask,
                                                 dout,
                                                 q,
                                                 k,
                                                 v,
                                                 out,
                                                 softmax_lse,
                                                 seed_offset,
                                                 *dq,
                                                 dk_tmp,
                                                 dv_tmp,
                                                 dropout,
                                                 causal,
                                                 batch_size,
                                                 max_seqlen_q,
                                                 num_heads,
                                                 head_size,
                                                 max_seqlen_k,
                                                 num_heads_k);

  VLOG(10) << "FlashAttn bwd seed: " << params.seed_offset_data[0]
           << ", offset: " << params.seed_offset_data[1];

  auto flash_cu_seqlens_q = DenseTensorToMcFlashAttnTensor(cu_seqlens_q);
  auto flash_cu_seqlens_k = DenseTensorToMcFlashAttnTensor(cu_seqlens_k);
  mcflashattnStatus_t succ =
      phi::dynload::mha_varlen_bwd(params.batch_size,
                                   total_q,
                                   params.num_heads,
                                   total_k,
                                   params.num_heads_k,
                                   head_size_og,
                                   params.dout,
                                   params.q,
                                   params.k,
                                   params.v,
                                   params.out,
                                   params.softmax_d,
                                   params.softmax_lse,
                                   params.dq,
                                   params.dk,
                                   params.dv,
                                   params.dq_accum,
                                   flash_cu_seqlens_q,
                                   flash_cu_seqlens_k,
                                   params.alibi_slopes,
                                   params.rng_state,
                                   params.seqlen_q,
                                   params.seqlen_k,
                                   params.p_dropout,
                                   params.softmax_scale,
                                   params.is_causal,
                                   params.window_size_left,
                                   params.window_size_right,
                                   params.deterministic,
                                   params.stream,
                                   params.extend_parameter);
  phi::dynload::release_tensor(cu_seqlens_q);
  phi::dynload::release_tensor(cu_seqlens_k);
  CheckFlashAttnStatus(succ);
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnGradKernel(const Context& ctx,
                         const DenseTensor& q,
                         const DenseTensor& k,
                         const DenseTensor& v,
                         const DenseTensor& out,
                         const DenseTensor& softmax_lse,
                         const DenseTensor& seed_offset,
                         const paddle::optional<DenseTensor>& attn_mask,
                         const DenseTensor& dout,
                         float dropout,
                         bool causal,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN

  ctx.template Alloc<T>(dq);
  DenseTensor dk_tmp;
  if (dk) {
    ctx.template Alloc<T>(dk);
    dk_tmp = *dk;
  } else {
    dk_tmp = EmptyLike<T, Context>(ctx, k);
  }

  DenseTensor dv_tmp;
  if (dv) {
    ctx.template Alloc<T>(dv);
    dv_tmp = *dv;
  } else {
    dv_tmp = EmptyLike<T, Context>(ctx, v);
  }
  // q, k, v [batch_size, seq_len, num_heads, head_dim]
  const auto& dims = q.dims();
  PADDLE_ENFORCE_EQ(dims.size(),
                    4,
                    phi::errors::InvalidArgument(
                        "flash_attn receive input with dim "
                        "[batch_size, seq_len, num_heads, head_dim]"));
  const int64_t batch_size = dims[0];
  const int64_t seqlen_q = dims[1];
  const int64_t num_heads = dims[2];
  const int64_t head_size = dims[3];
  const int64_t seqlen_k = k.dims()[1];
  const int64_t num_heads_k = k.dims()[2];
  const int64_t head_size_og = dout.dims()[3];
  // TODO(umiswing): add shape check
  PADDLE_ENFORCE_EQ(
      head_size_og,
      head_size,
      phi::errors::InvalidArgument(
          "flash_attn_bwd receive input with head_size_og == head_size"));

  FlashAttnParamsBwd params = FlashAttnParamsBwd(ctx,
                                                 attn_mask,
                                                 dout,
                                                 q,
                                                 k,
                                                 v,
                                                 out,
                                                 softmax_lse,
                                                 seed_offset,
                                                 *dq,
                                                 dk_tmp,
                                                 dv_tmp,
                                                 dropout,
                                                 causal,
                                                 batch_size,
                                                 seqlen_q,
                                                 num_heads,
                                                 head_size,
                                                 seqlen_k,
                                                 num_heads_k);

  VLOG(10) << "[FlashAttn Forward] q.shape=[" << q.dims() << "], k.shape=["
           << k.dims() << "], v.shape=[" << v.dims() << "]";
  VLOG(10) << "[FlashAttn Forward] dropout=" << dropout
           << ", seed=" << params.seed_offset_data[0]
           << ", offset=" << params.seed_offset_data[1];
  VLOG(10) << "[FlashAttn Forward] softmax_scale=" << params.softmax_scale;
  if (attn_mask.get_ptr()) {
    VLOG(10) << "[FlashAttn Backward] attn_mask.shape=["
             << (attn_mask.get_ptr())->dims() << "]";
  }
  mcflashattnStatus_t succ = phi::dynload::mha_bwd(params.batch_size,
                                                   params.seqlen_q,
                                                   params.num_heads,
                                                   params.seqlen_k,
                                                   params.num_heads_k,
                                                   params.head_size,
                                                   params.dout,
                                                   params.q,
                                                   params.k,
                                                   params.v,
                                                   params.out,
                                                   params.softmax_d,
                                                   params.softmax_lse,
                                                   params.dq,
                                                   params.dk,
                                                   params.dv,
                                                   params.dq_accum,
                                                   params.alibi_slopes,
                                                   params.attn_mask,
                                                   params.rng_state,
                                                   params.p_dropout,
                                                   params.softmax_scale,
                                                   params.is_causal,
                                                   params.window_size_left,
                                                   params.window_size_right,
                                                   params.deterministic,
                                                   params.stream,
                                                   params.extend_parameter);

  CheckFlashAttnStatus(succ);
#else
  RaiseNotSupportedError();
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(7).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}
