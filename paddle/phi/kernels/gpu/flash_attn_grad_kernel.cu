#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_grad_kernel.h"
#include "glog/logging.h"  // For VLOG()
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/gpu/flash_attn_utils.h"
#include "paddle/phi/kernels/reshape_kernel.h"

DECLARE_bool(cudnn_deterministic);

namespace phi {

template <typename T, typename Context>
void FlashAttnUnpaddedGradImpl(const Context& ctx,
                               const DenseTensor& q,
                               const DenseTensor& k,
                               const DenseTensor& v,
                               const DenseTensor& cu_seqlens_q,
                               const DenseTensor& cu_seqlens_k,
                               const DenseTensor& out,
                               const DenseTensor& softmax_lse,
                               const DenseTensor& seed_offset,
                               const paddle::optional<DenseTensor>& attn_mask,
                               const DenseTensor& dout,
                               int64_t max_seqlen_q,
                               int64_t max_seqlen_k,
                               float scale,
                               float dropout,
                               bool causal,
                               DenseTensor* dq,
                               DenseTensor* dk,
                               DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  const hipStream_t stream = ctx.stream();

  auto dims = q.dims();
  int64_t total_q = dims[0];
  int64_t num_heads = dims[1];
  int64_t head_size = dims[2];

  int64_t total_k = k.dims()[0];
  int64_t batch_size = cu_seqlens_q.numel() - 1;

  PADDLE_ENFORCE_NE(causal,
                    true,
                    phi::errors::InvalidArgument(
                        "attn_mask is not nullptr, causal can not be true"));

  PADDLE_ENFORCE_EQ(
      head_size == 32 || head_size == 64 || head_size == 128,
      true,
      phi::errors::InvalidArgument("The head_dim is expected to be either 32, "
                                   "64, or 128, but recieved %d.",
                                   head_size));
  const int64_t* seed_offset_data = seed_offset.data<int64_t>();
  uint64_t seed = static_cast<uint64_t>(seed_offset_data[0]);
  uint64_t offset = static_cast<uint64_t>(seed_offset_data[1]);
  VLOG(10) << "FlashAttn bwd seed: " << seed << ", offset: " << offset;

  int64_t seqlen_q = ((max_seqlen_q + 16 - 1) / 16) * 16;
  DenseTensor dsoftmax = Empty<float>(ctx, {batch_size, num_heads, seqlen_q});

  const DenseTensor* attn_mask_tensor = attn_mask.get_ptr();
  std::vector<int64_t> mask_dims = GetAttnMaskDims(attn_mask_tensor);

  int fa_num_splits = 0;
  bool fa_is_bf16 = q.dtype() == DataType::BFLOAT16;
  float fa_with_mask_scale = 1.0f;
  bool fa_zero_tensors = false;

  uint64_t workspace_size;

  int64_t q_size = total_q * num_heads * head_size;
  DenseTensor scaled_q = Empty<T>(ctx, {total_q, num_heads, head_size});
  ComputeScaleQ(ctx, q_size, scale, q.data<T>(), scaled_q.data<T>());

  bool succ = phi::dynload::flash_attn_bwd_with_bias_and_mask(
      static_cast<const void*>(scaled_q.data<T>()),
      static_cast<const void*>(k.data()),
      static_cast<const void*>(v.data()),
      static_cast<void*>(dq->data()),
      static_cast<void*>(dk->data()),
      static_cast<void*>(dv->data()),
      nullptr,  // set out to nullptr to calculate workspace size
      dout.data(),
      static_cast<const int32_t*>(cu_seqlens_q.data()),
      static_cast<const int32_t*>(cu_seqlens_k.data()),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      max_seqlen_q,
      max_seqlen_k,
      dropout,
      fa_with_mask_scale,
      fa_zero_tensors,
      fa_is_bf16,
      fa_num_splits,
      static_cast<const void*>(softmax_lse.data()),
      static_cast<void*>(dsoftmax.data()),
      nullptr,
      nullptr,
      &workspace_size,
      stream,
      seed,
      offset,
      attn_mask_tensor ? attn_mask_tensor->data() : nullptr,
      nullptr,
      mask_dims.data() ? mask_dims.data() : nullptr,
      nullptr);
  CheckFlashAttnStatus(succ);
  DenseTensor workspace;
  if (workspace_size > 0) {
    workspace = Empty<float>(
        ctx, {static_cast<int64_t>(workspace_size / sizeof(float))});
  }

  succ = phi::dynload::flash_attn_bwd_with_bias_and_mask(
      static_cast<const void*>(scaled_q.data<T>()),
      static_cast<const void*>(k.data()),
      static_cast<const void*>(v.data()),
      static_cast<void*>(dq->data()),
      static_cast<void*>(dk->data()),
      static_cast<void*>(dv->data()),
      out.data(),  // set out to nullptr to calculate workspace size
      dout.data(),
      static_cast<const int32_t*>(cu_seqlens_q.data()),
      static_cast<const int32_t*>(cu_seqlens_k.data()),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      max_seqlen_q,
      max_seqlen_k,
      dropout,
      fa_with_mask_scale,
      fa_zero_tensors,
      fa_is_bf16,
      fa_num_splits,
      static_cast<const void*>(softmax_lse.data()),
      static_cast<void*>(dsoftmax.data()),
      nullptr,
      workspace_size > 0 ? workspace.data() : nullptr,
      &workspace_size,
      stream,
      seed,
      offset,
      attn_mask_tensor ? attn_mask_tensor->data() : nullptr,
      nullptr,
      mask_dims.data() ? mask_dims.data() : nullptr,
      nullptr);
  CheckFlashAttnStatus(succ);

  ComputeScaleQ(ctx, q_size, scale, dq->data<T>(), dq->data<T>());
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnUnpaddedGradKernel(const Context& ctx,
                                 const DenseTensor& q,
                                 const DenseTensor& k,
                                 const DenseTensor& v,
                                 const DenseTensor& cu_seqlens_q,
                                 const DenseTensor& cu_seqlens_k,
                                 const DenseTensor& out,
                                 const DenseTensor& softmax_lse,
                                 const DenseTensor& seed_offset,
                                 const paddle::optional<DenseTensor>& attn_mask,
                                 const DenseTensor& dout,
                                 int64_t max_seqlen_q,
                                 int64_t max_seqlen_k,
                                 float scale,
                                 float dropout,
                                 bool causal,
                                 DenseTensor* dq,
                                 DenseTensor* dk,
                                 DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(dq);
  ctx.template Alloc<T>(dk);
  ctx.template Alloc<T>(dv);

  const hipStream_t stream = ctx.stream();

  // q,k,v [total_*, num_heads, head_dim]
  auto dims = q.dims();

  if (attn_mask.get_ptr()) {
    FlashAttnUnpaddedGradImpl<T, Context>(ctx,
                                          q,
                                          k,
                                          v,
                                          cu_seqlens_q,
                                          cu_seqlens_k,
                                          out,
                                          softmax_lse,
                                          seed_offset,
                                          attn_mask,
                                          dout,
                                          max_seqlen_q,
                                          max_seqlen_k,
                                          scale,
                                          dropout,
                                          causal,
                                          dq,
                                          dk,
                                          dv);
  } else {
    const int64_t total_q = dims[0];
    const int64_t batch_size = cu_seqlens_q.numel() - 1;
    const int64_t num_heads = dims[1];
    const int64_t head_size_og = dout.dims()[2];
    const int64_t head_size = dims[2];
    const int64_t total_k = k.dims()[0];
    const int64_t num_heads_k = k.dims()[1];

    // TODO(umiswing): add deterministic in fa2.
    // int num_splits = 0;  // 0 for an internal heuristic, which is optimal
    // if (FLAGS_cudnn_deterministic) {
    //   num_splits = 1;
    // }

    // TODO(umiswing): add shape check
    PADDLE_ENFORCE_EQ(
        head_size_og,
        head_size,
        phi::errors::InvalidArgument(
            "flash_attn_bwd receive input with head_size_og == head_size"));

    FlashAttnBwdParamsV2 params =
        FlashAttnBwdParamsV2(ctx,
                             batch_size,
                             max_seqlen_q,
                             max_seqlen_k,
                             num_heads,
                             num_heads_k,
                             head_size,
                             dropout,
                             scale,
                             causal,
                             q.dtype(),
                             seed_offset.data<int64_t>());

    VLOG(10) << "FlashAttn bwd seed: " << params.seed
             << ", offset: " << params.offset;

    bool succ =
        phi::dynload::flash_attn_varlen_bwd(dout.data(),
                                            q.data(),
                                            k.data(),
                                            v.data(),
                                            out.data(),
                                            params.softmax_d.data(),
                                            softmax_lse.data(),
                                            cu_seqlens_q.data<int32_t>(),
                                            cu_seqlens_k.data<int32_t>(),
                                            params.rng_state.data(),
                                            dq->data(),
                                            dk->data(),
                                            dv->data(),
                                            params.dq_accum.data(),
                                            params.batch_size,
                                            params.max_seqlen_q,
                                            params.max_seqlen_k,
                                            params.seqlen_q_rounded,
                                            params.seqlen_k_rounded,
                                            params.num_heads,
                                            params.num_heads_k,
                                            params.head_size,
                                            params.head_size_rounded,
                                            params.dropout,
                                            params.scale,
                                            params.causal,
                                            params.is_bf16,
                                            stream,
                                            params.seed,
                                            params.offset);
    CheckFlashAttnStatus(succ);
  }
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnGradKernel(const Context& ctx,
                         const DenseTensor& q,
                         const DenseTensor& k,
                         const DenseTensor& v,
                         const DenseTensor& out,
                         const DenseTensor& softmax_lse,
                         const DenseTensor& seed_offset,
                         const paddle::optional<DenseTensor>& attn_mask,
                         const DenseTensor& dout,
                         float dropout,
                         bool causal,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [batch_size, seq_len, num_heads, head_dim]

  const auto& dims = q.dims();
  const int64_t batch_size = dims[0];
  const int64_t seqlen_q = dims[1];
  const int64_t num_heads = dims[2];
  const int64_t head_size_og = dout.dims()[3];
  const int64_t head_size = dims[3];
  const int64_t seqlen_k = k.dims()[1];
  const int64_t num_heads_k = k.dims()[2];

  const int64_t total_q = batch_size * seqlen_q;
  const int64_t total_k = batch_size * seqlen_k;

  // TODO(umiswing): add shape check
  PADDLE_ENFORCE_EQ(
      head_size_og,
      head_size,
      phi::errors::InvalidArgument(
          "flash_attn_bwd receive input with head_size_og == head_size"));

  VLOG(10) << "FlashAttn bwd dims q[" << q.dims() << "], k[" << k.dims()
           << "], v[" << v.dims() << "]";

  const float scale = 1.0f / std::sqrt(head_size);
  if (attn_mask.get_ptr()) {
    DenseTensor q_t_s, k_t_s, v_t_s;
    q_t_s.ShareDataWith(q).Resize({total_q, num_heads, head_size});
    k_t_s.ShareDataWith(k).Resize({total_k, num_heads, head_size});
    v_t_s.ShareDataWith(v).Resize({total_k, num_heads, head_size});

    DenseTensor cu_seqlens_q;
    DenseTensor cu_seqlens_k;
    ArangeNullaryKernel<int32_t, Context>(
        ctx, 0, (batch_size + 1) * seqlen_q, seqlen_q, &cu_seqlens_q);
    ArangeNullaryKernel<int32_t, Context>(
        ctx, 0, (batch_size + 1) * seqlen_k, seqlen_k, &cu_seqlens_k);

    FlashAttnUnpaddedGradKernel<T, Context>(ctx,
                                            q_t_s,
                                            k_t_s,
                                            v_t_s,
                                            cu_seqlens_q,
                                            cu_seqlens_k,
                                            out,
                                            softmax_lse,
                                            seed_offset,
                                            attn_mask,
                                            dout,
                                            seqlen_q,
                                            seqlen_k,
                                            scale,
                                            dropout,
                                            causal,
                                            dq,
                                            dk,
                                            dv);
  } else {
    FlashAttnBwdParamsV2 params =
        FlashAttnBwdParamsV2(ctx,
                             batch_size,
                             seqlen_q,
                             seqlen_k,
                             num_heads,
                             num_heads_k,
                             head_size,
                             dropout,
                             scale,
                             causal,
                             q.dtype(),
                             seed_offset.data<int64_t>());

    ctx.template Alloc<T>(dq);
    ctx.template Alloc<T>(dk);
    ctx.template Alloc<T>(dv);

    hipStream_t stream = ctx.stream();

    VLOG(10) << "FlashAttn bwd seed: " << params.seed
             << ", offset: " << params.offset;

    bool succ = phi::dynload::flash_attn_bwd(dout.data(),
                                             q.data(),
                                             k.data(),
                                             v.data(),
                                             out.data(),
                                             params.softmax_d.data(),
                                             softmax_lse.data(),
                                             params.rng_state.data(),
                                             dq->data(),
                                             dk->data(),
                                             dv->data(),
                                             params.dq_accum.data(),
                                             params.batch_size,
                                             params.max_seqlen_q,
                                             params.max_seqlen_k,
                                             params.seqlen_q_rounded,
                                             params.seqlen_k_rounded,
                                             params.num_heads,
                                             params.num_heads_k,
                                             params.head_size,
                                             params.head_size_rounded,
                                             params.dropout,
                                             params.scale,
                                             params.causal,
                                             params.is_bf16,
                                             stream,
                                             params.seed,
                                             params.offset);
    CheckFlashAttnStatus(succ);
  }
#else
  RaiseNotSupportedError();
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(7).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}
