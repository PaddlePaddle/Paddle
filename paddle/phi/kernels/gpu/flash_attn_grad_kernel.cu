#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_grad_kernel.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/reshape_kernel.h"

#ifdef PADDLE_WITH_FLASHATTN
#include "paddle/phi/backends/dynload/flashattn.h"
#endif

namespace phi {

template <typename T, typename Context>
void FlashAttnUnpaddedGradKernel(const Context& ctx,
                                 const DenseTensor& q,
                                 const DenseTensor& k,
                                 const DenseTensor& v,
                                 const DenseTensor& cu_seqlens_q,
                                 const DenseTensor& cu_seqlens_k,
                                 const DenseTensor& out,
                                 const DenseTensor& softmax_lse,
                                 const DenseTensor& seed_offset,
                                 const DenseTensor& dout,
                                 int64_t max_seqlen_q,
                                 int64_t max_seqlen_k,
                                 float scale,
                                 float dropout,
                                 bool causal,
                                 DenseTensor* dq,
                                 DenseTensor* dk,
                                 DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(dq);
  ctx.template Alloc<T>(dk);
  ctx.template Alloc<T>(dv);

  hipStream_t stream = ctx.stream();
  bool is_bf16 = q.dtype() == DataType::BFLOAT16 ? true : false;

  // q,k,v [total_*, num_heads, head_dim]

  auto dims = q.dims();
  int64_t total_q = dims[0];
  int64_t num_heads = dims[1];
  int64_t head_size = dims[2];

  int64_t total_k = k.dims()[0];
  int64_t batch_size = cu_seqlens_q.numel() - 1;

  int num_splits = 0;  // 0 for an internal heuristic, which is optimal
  bool zero_tensors = false;

  std::vector<int64_t> seed_offset_vec;
  phi::TensorToVector<int64_t>(seed_offset, ctx, &seed_offset_vec);
  uint64_t seed = seed_offset_vec[0];
  uint64_t offset = seed_offset_vec[1];

  int64_t seq_len_q = ((max_seqlen_q + 16 - 1) / 16) * 16;
  DenseTensor dsoftmax = Empty<float>(ctx, {batch_size, num_heads, seq_len_q});

  uint64_t workspace_size;

  // calculate workspace size before execution
  bool succ = phi::dynload::flash_attn_bwd(
      q.data(),
      k.data(),
      v.data(),
      dq->data(),
      dk->data(),
      dv->data(),
      nullptr,  // for calculation workspace size
      dout.data(),
      cu_seqlens_q.data(),
      cu_seqlens_k.data(),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      max_seqlen_q,
      max_seqlen_k,
      dropout,
      scale,
      zero_tensors,
      causal,
      is_bf16,
      num_splits,
      const_cast<float*>(softmax_lse.data<float>()),
      dsoftmax.data(),
      nullptr,
      &workspace_size,
      stream,
      seed,
      offset);

  if (!succ) {
    PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
  }

  DenseTensor workspace;
  if (workspace_size > 0) {
    workspace = Empty<float>(ctx, {int64_t(workspace_size / sizeof(float))});
  }

  succ = phi::dynload::flash_attn_bwd(
      q.data(),
      k.data(),
      v.data(),
      dq->data(),
      dk->data(),
      dv->data(),
      out.data(),
      dout.data(),
      cu_seqlens_q.data(),
      cu_seqlens_k.data(),
      total_q,
      total_k,
      batch_size,
      num_heads,
      head_size,
      max_seqlen_q,
      max_seqlen_k,
      dropout,
      scale,
      zero_tensors,
      causal,
      is_bf16,
      num_splits,
      const_cast<float*>(softmax_lse.data<float>()),
      dsoftmax.data(),
      workspace_size > 0 ? workspace.data() : nullptr,
      &workspace_size,
      stream,
      seed,
      offset);

  if (!succ) {
    PADDLE_THROW(phi::errors::External(phi::dynload::flash_attn_error()));
  }

#endif
}

template <typename T, typename Context>
void FlashAttnGradKernel(const Context& ctx,
                         const DenseTensor& q,
                         const DenseTensor& k,
                         const DenseTensor& v,
                         const DenseTensor& out,
                         const DenseTensor& softmax_lse,
                         const DenseTensor& seed_offset,
                         const DenseTensor& dout,
                         float dropout,
                         bool causal,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [batch_size, seq_len, num_heads, head_dim]

  auto dims = q.dims();
  int64_t batch_size = dims[0];
  int64_t seq_len_q = dims[1];
  int64_t num_heads = dims[2];
  int64_t head_size = dims[3];

  int64_t seq_len_k = k.dims()[1];

  int64_t total_q = batch_size * seq_len_q;
  int64_t total_k = batch_size * seq_len_k;

  float scale = 1.0f / std::sqrt(head_size);

  DenseTensor q_t_s =
      Reshape<T, Context>(ctx, q, {total_q, num_heads, head_size});
  DenseTensor k_t_s =
      Reshape<T, Context>(ctx, k, {total_k, num_heads, head_size});
  DenseTensor v_t_s =
      Reshape<T, Context>(ctx, v, {total_k, num_heads, head_size});

  DenseTensor cu_seqlens_q;
  DenseTensor cu_seqlens_k;
  ArangeNullaryKernel<int32_t, Context>(
      ctx, 0, (batch_size + 1) * seq_len_q, seq_len_q, &cu_seqlens_q);
  ArangeNullaryKernel<int32_t, Context>(
      ctx, 0, (batch_size + 1) * seq_len_k, seq_len_k, &cu_seqlens_k);

  FlashAttnUnpaddedGradKernel<T, Context>(ctx,
                                          q_t_s,
                                          k_t_s,
                                          v_t_s,
                                          cu_seqlens_q,
                                          cu_seqlens_k,
                                          out,
                                          softmax_lse,
                                          seed_offset,
                                          dout,
                                          seq_len_q,
                                          seq_len_k,
                                          scale,
                                          dropout,
                                          causal,
                                          dq,
                                          dk,
                                          dv);

#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(7).SetBackend(phi::Backend::CPU);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(phi::Backend::CPU);  // seed_offset
}
