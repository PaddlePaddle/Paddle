#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_grad_kernel.h"
#include <cstddef>
#include "glog/logging.h"  // For VLOG()
#include "paddle/common/enforce.h"
#include "paddle/common/flags.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"
#include "paddle/phi/kernels/gpu/flash_attn_utils.h"
#include "paddle/phi/kernels/reduce_sum_kernel.h"

COMMON_DECLARE_bool(cudnn_deterministic);

namespace phi {

int get_num_split() {
  // 0 for an internal heuristic, which is optimal
  return FLAGS_cudnn_deterministic ? 1 : 0;
}

template <typename T, uint64_t HeaddimDiv32>
static __global__ void SumStridedKV(const T* src,
                                    T* dst,
                                    const uint64_t sRowDim1,
                                    const uint64_t sRowDim2,
                                    const uint64_t sRowDim3,
                                    const uint64_t sColDim,
                                    const uint64_t sRowStride1,
                                    const uint64_t sRowStride2,
                                    const uint64_t sColStride,
                                    const uint64_t dRowStride1,
                                    const uint64_t dRowStride2) {
  // SrcShape [seqlen, num_heads_k, num_heads/num_heads_k, headdim]
  // AxisName [row1  , row2       , col                  , row3   ]
  // LoopMap  [blockx, thready    , serialreduce         , threadx]
  // Ensure blockDim.x == 32 && blockDim.z == 1
  // Ensure sRowStride3 == dRowStride3 == 1 (headdim dim is contiguous)
  using IndexType = uint64_t;
  constexpr IndexType BlockDimX = 32;
  const IndexType SRow1Begin = blockIdx.x * sRowStride1;
  const IndexType SRow1End = sRowDim1 * sRowStride1;
  const IndexType SRow1Stride = gridDim.x * sRowStride1;

  const IndexType SRow2Begin = threadIdx.y * sRowStride2;
  const IndexType SRow2End = sRowDim2 * sRowStride2;
  const IndexType SRow2Stride = blockDim.y * sRowStride2;

  // const IndexType SRow3Begin = threadIdx.x * sRowStride3;
  // const IndexType SRow3End = sRowDim3 * sRowStride3;
  // const IndexType SRow3Stride = BlockDimX * sRowStride3;

  constexpr IndexType SColBegin = 0;
  const IndexType SColEnd = sColDim * sColStride;
  const IndexType SColStride = sColStride;

  const IndexType DRow1Begin = blockIdx.x * dRowStride1;
  const IndexType DRow1Stride = gridDim.x * dRowStride1;

  const IndexType DRow2Begin = threadIdx.y * dRowStride2;
  const IndexType DRow2Stride = dRowStride2;

  // const IndexType DRow3Begin = threadIdx.x * dRowStride3;
  // const IndexType DRow3Stride = blockDim.x * dRowStride3;

  for (auto row1 = SRow1Begin, drow1 = DRow1Begin; row1 < SRow1End;
       row1 += SRow1Stride, drow1 += DRow1Stride) {
    for (auto row2 = SRow2Begin, drow2 = DRow2Begin; row2 < SRow2End;
         row2 += SRow2Stride, drow2 += DRow2Stride) {
      const auto i1 = row1 + row2 + threadIdx.x;
      const auto di1 = drow1 + drow2 + threadIdx.x;
      T v[HeaddimDiv32];
#pragma unroll
      for (auto i = IndexType(0); i < HeaddimDiv32; i++) {
        v[i] = T{0};
      }
      for (auto col = SColBegin; col < SColEnd; col += SColStride) {
        const auto i2 = i1 + col;
#pragma unroll
        for (auto i = IndexType(0); i < HeaddimDiv32; i++) {
          v[i] += src[i2 + i * BlockDimX];
        }
      }
#pragma unroll
      for (auto i = IndexType(0); i < HeaddimDiv32; i++) {
        dst[di1 + i * BlockDimX] = v[i];
      }
    }
  }
}

template <typename T>
static auto selectSumkernel(int64_t headdim) {
  PADDLE_ENFORCE_LE(headdim, 256, "FlashAttention only support headdim <= 256");
  PADDLE_ENFORCE_EQ(
      headdim % 32, 0, "FlashAttention only support headdim %% 32 == 0");
  PADDLE_ENFORCE_NE(headdim, 0, "Headdim can't be zero");
#define CASEN(n) \
  case n:        \
    return SumStridedKV<T, n>;
  switch (headdim / 32) {
    CASEN(1);
    CASEN(2);
    CASEN(3);
    CASEN(4);
    CASEN(5);
    CASEN(6);
    CASEN(7);
    CASEN(8);
  }
  PADDLE_FATAL("Unreachable in selectSumKernel");
#undef CASEN
}

template <typename T, typename Context>
static void kvReduceForGQA(const Context& ctx,
                           const DenseTensor& dk_tmp,
                           DenseTensor* dk) {
  PADDLE_ENFORCE_EQ(
      dk->strides()[2], 1, "headdim dimention must be contiguous");
  PADDLE_ENFORCE_EQ(
      dk_tmp.strides()[3], 1, "headdim dimention must be contiguous");
  const int64_t reduceDimSize = dk_tmp.dims()[2];
  const size_t blockNum =
      std::min((static_cast<int64_t>(dk_tmp.dims()[0] + 31) / 32),
               static_cast<int64_t>(1024l));
  const dim3 threadNum{32, 4, 1};
  auto sumkernel = selectSumkernel<T>(dk_tmp.dims()[3]);
  sumkernel<<<blockNum, threadNum, 0, ctx.stream()>>>(
      reinterpret_cast<const T*>(dk_tmp.data()),
      reinterpret_cast<T*>(dk->data()),
      dk_tmp.dims()[0],
      dk_tmp.dims()[1],
      dk_tmp.dims()[3],
      dk_tmp.dims()[2],
      dk_tmp.strides()[0],
      dk_tmp.strides()[1],
      // dk_tmp.strides()[3],
      dk_tmp.strides()[2],
      dk->strides()[0],
      dk->strides()[1]
      // dk->strides()[2]
  );
}
template <typename T, typename Context>
static void kvReduceBatchedForGQA(const Context& ctx,
                                  const DenseTensor& dk_tmp,
                                  DenseTensor* dk) {
  PADDLE_ENFORCE_EQ(
      dk->strides()[3], 1, "headdim dimention must be contiguous");
  PADDLE_ENFORCE_EQ(
      dk_tmp.strides()[4], 1, "headdim dimention must be contiguous");
  PADDLE_ENFORCE_EQ(dk->strides()[0],
                    dk->strides()[1] * dk->dims()[1],
                    "batchsize dimention must be contiguous");
  PADDLE_ENFORCE_EQ(dk_tmp.strides()[0],
                    dk_tmp.strides()[1] * dk_tmp.dims()[1],
                    "batchsize dimention must be contiguous");
  const int64_t reduceDimSize = dk_tmp.dims()[3];
  const size_t blockNum = std::min(
      (static_cast<int64_t>(dk_tmp.dims()[0] * dk_tmp.dims()[1] + 31) / 32),
      static_cast<int64_t>(1024l));
  const dim3 threadNum{32, 4, 1};
  auto sumkernel = selectSumkernel<T>(dk_tmp.dims()[4]);
  // here implicitly flat [batch,seqlen], and require batch dim to be contiguous
  sumkernel<<<blockNum, threadNum, 0, ctx.stream()>>>(
      reinterpret_cast<const T*>(dk_tmp.data()),
      reinterpret_cast<T*>(dk->data()),
      dk_tmp.dims()[0] * dk_tmp.dims()[1],
      dk_tmp.dims()[2],
      dk_tmp.dims()[4],
      dk_tmp.dims()[3],
      dk_tmp.strides()[1],
      dk_tmp.strides()[2],
      // dk_tmp.strides()[4],
      dk_tmp.strides()[3],
      dk->strides()[1],
      dk->strides()[2]
      // dk->strides()[3]
  );
}

template <typename T, typename Context>
void FlashAttnUnpaddedGradBaseKernel(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& cu_seqlens_q,
    const DenseTensor& cu_seqlens_k,
    const DenseTensor& out,
    const DenseTensor& softmax_lse,
    const DenseTensor& seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    const DenseTensor& dout,
    int64_t max_seqlen_q,
    int64_t max_seqlen_k,
    float scale,
    float dropout,
    bool causal,
    DenseTensor* dq,
    DenseTensor* dk,
    DenseTensor* dv,
    bool varlen_padded) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [total_*, num_heads, head_dim]
  auto dims = q.dims();

  const int64_t batch_size = cu_seqlens_q.numel() - 1;
  const int64_t num_heads = dims[1];
  const int64_t head_size_og = dout.dims()[2];
  const int64_t head_size = dims[2];
  const int64_t total_k = k.dims()[0];
  const int64_t num_heads_k = k.dims()[1];

  bool is_mha = (num_heads == num_heads_k);

  DenseTensor* kdq = dq;
  DenseTensor dq_tmp;
  if (!dq) {
    dq_tmp.Resize(dims);
    ctx.template Alloc<T>(&dq_tmp);
    kdq = &dq_tmp;
  }

  std::initializer_list<int64_t> dk_dv_shape = {
      total_k, num_heads_k, num_heads / num_heads_k, head_size};

  DenseTensor *kdk = dk, *kdv = dv;
  DenseTensor dk_tmp;
  if (!dk || !is_mha) {
    dk_tmp.Resize(dk_dv_shape);
    ctx.template Alloc<T>(&dk_tmp);
    kdk = &dk_tmp;
  }

  DenseTensor dv_tmp;
  if (!dv || !is_mha) {
    dv_tmp.Resize(dk_dv_shape);
    ctx.template Alloc<T>(&dv_tmp);
    kdv = &dv_tmp;
  }

  const hipStream_t stream = ctx.stream();

  int num_splits = get_num_split();

  // TODO(umiswing): add shape check
  PADDLE_ENFORCE_EQ(
      head_size_og,
      head_size,
      phi::errors::InvalidArgument(
          "flash_attn_bwd receive input with head_size_og == head_size"));

  FlashAttnBwdParamsV2 params =
      FlashAttnBwdParamsV2(ctx,
                           batch_size,
                           max_seqlen_q,
                           max_seqlen_k,
                           num_heads,
                           num_heads_k,
                           head_size,
                           dropout,
                           scale,
                           causal,
                           0,  // attn_mask_start_row
                           q.dtype(),
                           attn_mask,
                           nullptr,  // attn_mask_start_row_indices,
                           nullptr,  // attn_mask_end_row_indices,
                           seed_offset.data<int64_t>());

  VLOG(10) << "FlashAttn bwd seed: " << params.seed
           << ", offset: " << params.offset;

  bool succ = phi::dynload::flash_attn_varlen_bwd(
      dout.data(),
      q.data(),
      k.data(),
      v.data(),
      out.data(),
      params.softmax_d.data(),
      softmax_lse.data(),
      cu_seqlens_q.data<int32_t>(),
      cu_seqlens_k.data<int32_t>(),
      params.rng_state.data(),
      kdq->data(),
      kdk->data(),
      kdv->data(),
      params.dq_accum.data(),
      params.batch_size,
      params.max_seqlen_q,
      params.max_seqlen_k,
      params.seqlen_q_rounded,
      params.seqlen_k_rounded,
      params.num_heads,
      params.num_heads_k,
      params.head_size,
      params.head_size_rounded,
      params.dropout,
      params.softmax_scale,
      1.0f / params.softmax_scale,
      params.causal,
      params.is_bf16,
      num_splits,
      stream,
      params.seed,
      params.offset,
      params.attn_mask_tensor ? params.attn_mask_tensor->data() : nullptr,
      params.attn_mask_tensor ? params.mask_dims.data() : nullptr,
      q.strides()[0],
      k.strides()[0],
      v.strides()[0],
      q.strides()[1],
      k.strides()[1],
      v.strides()[1],
      out.strides()[0],
      out.strides()[1],
      max_seqlen_q * q.strides()[0],
      max_seqlen_k * k.strides()[0],
      max_seqlen_k * v.strides()[0],
      max_seqlen_q * out.strides()[0],
      kdq->strides()[0],
      kdk->strides()[0],
      kdv->strides()[0],
      kdq->strides()[1],
      kdk->strides()[kdk->strides().size() - 2],
      kdv->strides()[kdv->strides().size() - 2],
      dout.strides()[0],
      dout.strides()[1],
      max_seqlen_q * kdq->strides()[0],
      max_seqlen_k * kdk->strides()[0],
      max_seqlen_k * kdv->strides()[0],
      max_seqlen_q * dout.strides()[0],
      varlen_padded);
  CheckFlashAttnStatus(succ);
  if (!is_mha) {
    if (dk) {
      if (dk->meta().is_contiguous())
        phi::SumKernel<T, Context>(ctx, dk_tmp, {2}, dk->type(), false, dk);
      else
        kvReduceForGQA<T, Context>(ctx, dk_tmp, dk);
    }
    if (dv) {
      if (dv->meta().is_contiguous())
        phi::SumKernel<T, Context>(ctx, dv_tmp, {2}, dv->type(), false, dv);
      else
        kvReduceForGQA<T, Context>(ctx, dv_tmp, dv);
    }
  }
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnUnpaddedGradKernel(const Context& ctx,
                                 const DenseTensor& q,
                                 const DenseTensor& k,
                                 const DenseTensor& v,
                                 const DenseTensor& cu_seqlens_q,
                                 const DenseTensor& cu_seqlens_k,
                                 const DenseTensor& out,
                                 const DenseTensor& softmax_lse,
                                 const DenseTensor& seed_offset,
                                 const paddle::optional<DenseTensor>& attn_mask,
                                 const DenseTensor& dout,
                                 int64_t max_seqlen_q,
                                 int64_t max_seqlen_k,
                                 float scale,
                                 float dropout,
                                 bool causal,
                                 DenseTensor* dq,
                                 DenseTensor* dk,
                                 DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  if (dq) {
    ctx.template Alloc<T>(dq);
  }
  if (dk) {
    ctx.template Alloc<T>(dk);
  }
  if (dv) {
    ctx.template Alloc<T>(dv);
  }
  FlashAttnUnpaddedGradBaseKernel<T>(ctx,
                                     q,
                                     k,
                                     v,
                                     cu_seqlens_q,
                                     cu_seqlens_k,
                                     out,
                                     softmax_lse,
                                     seed_offset,
                                     attn_mask,
                                     dout,
                                     max_seqlen_q,
                                     max_seqlen_k,
                                     scale,
                                     dropout,
                                     causal,
                                     dq,
                                     dk,
                                     dv,
                                     false /*varlen_padded*/);
#else
  RaiseNotSupportedError();
#endif
}

static void sliceFlattenView(const DenseTensor& in,
                             DenseTensor* out,
                             int axis,
                             int64_t offset,
                             int64_t sliceLength) {
  PADDLE_ENFORCE_LT(
      axis,
      in.dims().size(),
      phi::errors::InvalidArgument("sliceView receive axis out of bound"));
  std::array<int64_t, DDim::kMaxRank> dimArr;
  std::array<int64_t, DDim::kMaxRank> strideArr;
  auto id = dimArr.begin(), is = strideArr.begin();
  for (int i = 0; i < in.dims().size(); i++) {
    if (i == axis) continue;
    if (i == axis + 1)
      *id = in.dims()[i] * sliceLength;
    else
      *id = in.dims()[i];
    *is = in.strides()[i];
    id++;
    is++;
  }
  *out = DenseTensor{
      in.Holder(),
      DenseTensorMeta{in.dtype(),
                      DDim{dimArr.data(), in.dims().size() - 1},
                      DDim(strideArr.data(), in.dims().size() - 1)}};
  out->set_offset(in.offset() +
                  offset * in.strides()[axis] * SizeOf(out->dtype()));
}
template <typename OutT>
struct ZeroFunctor {
  __device__ __forceinline__ OutT operator()() const {
    return static_cast<OutT>(0);
  }
};
template <typename T, typename Context>
void FlashAttnVarlenQKVPackedGradKernel(
    const Context& ctx,
    const DenseTensor& qkv,
    const DenseTensor& cu_seqlens_q,
    const DenseTensor& cu_seqlens_k,
    const DenseTensor& out,
    const DenseTensor& softmax_lse,
    const DenseTensor& seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    const DenseTensor& dout,
    int64_t max_seqlen_q,
    int64_t max_seqlen_k,
    float scale,
    float dropout,
    bool causal,
    bool varlen_padded,
    DenseTensor* dqkv) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [total_*, num_heads, head_dim]
  const auto head_groupnum = qkv.dims()[1];  // nheads/nheads_k + 1 + 1
  DenseTensor q, k, v;
  sliceFlattenView(qkv, &q, 1, 0, head_groupnum - 2);
  sliceFlattenView(qkv, &k, 1, head_groupnum - 2, 1);
  sliceFlattenView(qkv, &v, 1, head_groupnum - 1, 1);
  // DenseTensor dqkv_tmp;
  if (!dqkv) {
    return;
    // dqkv is the only output. No need to compute if no dqkv
    // dqkv_tmp.Resize(qkv.dims());
    // dqkv = &dqkv_tmp;
  }
  ctx.template Alloc<T>(dqkv);
  {
    std::vector<const DenseTensor*> inputs{};
    std::vector<DenseTensor*> outputs{dqkv};
    phi::funcs::ElementwiseKernel<T>(ctx, inputs, &outputs, ZeroFunctor<T>());
  }
  DenseTensor dq, dk, dv;
  sliceFlattenView(*dqkv, &dq, 1, 0, head_groupnum - 2);
  sliceFlattenView(*dqkv, &dk, 1, head_groupnum - 2, 1);
  sliceFlattenView(*dqkv, &dv, 1, head_groupnum - 1, 1);
  FlashAttnUnpaddedGradBaseKernel<T>(ctx,
                                     q,
                                     k,
                                     v,
                                     cu_seqlens_q,
                                     cu_seqlens_k,
                                     out,
                                     softmax_lse,
                                     seed_offset,
                                     attn_mask,
                                     dout,
                                     max_seqlen_q,
                                     max_seqlen_k,
                                     scale,
                                     dropout,
                                     causal,
                                     &dq,
                                     &dk,
                                     &dv,
                                     varlen_padded);
#else
  RaiseNotSupportedError();
#endif
}
template <typename T, typename Context>
void FlashAttnGradBaseKernel(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& out,
    const DenseTensor& softmax_lse,
    const DenseTensor& seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    const paddle::optional<DenseTensor>& attn_mask_start_row_indices,
    const paddle::optional<DenseTensor>& attn_mask_end_row_indices,
    const DenseTensor& dout,
    float dropout,
    bool causal,
    int attn_mask_start_row,
    DenseTensor* dq,
    DenseTensor* dk,
    DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  // q, k, v [batch_size, seq_len, num_heads, head_dim]
  const auto& dims = q.dims();

  const int64_t batch_size = dims[0];
  const int64_t seqlen_q = dims[1];
  const int64_t num_heads = dims[2];
  const int64_t head_size_og = dout.dims()[3];
  const int64_t head_size = dims[3];
  const int64_t seqlen_k = k.dims()[1];
  const int64_t num_heads_k = k.dims()[2];

  bool is_mha = (num_heads == num_heads_k);

  std::initializer_list<int64_t> dk_dv_shape = {
      batch_size, seqlen_k, num_heads_k, num_heads / num_heads_k, head_size};
  DenseTensor* kdq = dq;
  DenseTensor dq_tmp;
  if (!dq) {
    dq_tmp.Resize(dims);
    ctx.template Alloc<T>(&dq_tmp);
    kdq = &dq_tmp;
  }

  DenseTensor *kdk = dk, *kdv = dv;
  DenseTensor dk_tmp;
  if (!dk || !is_mha) {
    dk_tmp.Resize(dk_dv_shape);
    ctx.template Alloc<T>(&dk_tmp);
    kdk = &dk_tmp;
  }

  DenseTensor dv_tmp;
  if (!dv || !is_mha) {
    dv_tmp.Resize(dk_dv_shape);
    ctx.template Alloc<T>(&dv_tmp);
    kdv = &dv_tmp;
  }

  const hipStream_t stream = ctx.stream();

  // TODO(umiswing): add shape check
  PADDLE_ENFORCE_EQ(
      head_size_og,
      head_size,
      phi::errors::InvalidArgument(
          "flash_attn_bwd receive input with head_size_og == head_size"));

  const float softmax_scale = 1.0f / std::sqrt(head_size);
  const float softmax_unscale = std::sqrt(head_size);

  FlashAttnBwdParamsV2 params =
      FlashAttnBwdParamsV2(ctx,
                           batch_size,
                           seqlen_q,
                           seqlen_k,
                           num_heads,
                           num_heads_k,
                           head_size,
                           dropout,
                           softmax_scale,
                           causal,
                           attn_mask_start_row,
                           q.dtype(),
                           attn_mask,
                           attn_mask_start_row_indices,
                           attn_mask_end_row_indices,
                           seed_offset.data<int64_t>());

  VLOG(10) << "[FlashAttn Forward] q.shape=[" << q.dims() << "], k.shape=["
           << k.dims() << "], v.shape=[" << v.dims() << "]";
  VLOG(10) << "[FlashAttn Forward] dropout=" << dropout
           << ", seed=" << params.seed << ", offset=" << params.offset;
  VLOG(10) << "[FlashAttn Forward] softmax_scale=" << softmax_scale
           << ", softmax_unscale=" << softmax_unscale;
  if (attn_mask.get_ptr()) {
    VLOG(10) << "[FlashAttn Backward] attn_mask.shape=["
             << (attn_mask.get_ptr())->dims() << "]";
  }

  int num_splits = get_num_split();

  bool succ = phi::dynload::flash_attn_bwd(
      dout.data(),
      q.data(),
      k.data(),
      v.data(),
      out.data(),
      params.softmax_d.data(),
      softmax_lse.data(),
      params.rng_state.data(),
      kdq->data(),
      kdk->data(),
      kdv->data(),
      params.dq_accum.data(),
      params.batch_size,
      params.max_seqlen_q,
      params.max_seqlen_k,
      params.seqlen_q_rounded,
      params.seqlen_k_rounded,
      params.num_heads,
      params.num_heads_k,
      params.head_size,
      params.head_size_rounded,
      params.dropout,
      params.softmax_scale,
      softmax_unscale,
      params.causal,
      params.is_bf16,
      num_splits,
      stream,
      params.seed,
      params.offset,
      params.attn_mask_tensor ? params.attn_mask_tensor->data() : nullptr,
      params.attn_mask_tensor ? params.mask_dims.data() : nullptr,
      params.attn_mask_start_row_indices_tensor
          ? params.attn_mask_start_row_indices_tensor->data()
          : nullptr,
      params.attn_mask_start_row_indices_tensor
          ? params.attn_mask_start_row_indices_dims.data()
          : nullptr,
      params.attn_mask_end_row_indices_tensor
          ? params.attn_mask_end_row_indices_tensor->data()
          : nullptr,
      params.attn_mask_start_row,
      q.strides()[1],
      k.strides()[1],
      v.strides()[1],
      q.strides()[2],
      k.strides()[2],
      v.strides()[2],
      out.strides()[1],
      out.strides()[2],
      q.strides()[0],
      k.strides()[0],
      v.strides()[0],
      out.strides()[0],
      kdq->strides()[1],
      kdk->strides()[1],
      kdv->strides()[1],
      kdq->strides()[2],
      kdk->strides()[kdk->strides().size() - 2],
      kdv->strides()[kdv->strides().size() - 2],
      dout.strides()[1],
      dout.strides()[2],
      kdq->strides()[0],
      kdk->strides()[0],
      kdv->strides()[0],
      dout.strides()[0]);
  CheckFlashAttnStatus(succ);
  if (!is_mha) {
    if (dk) {
      if (dk->meta().is_contiguous())
        phi::SumKernel<T, Context>(ctx, dk_tmp, {3}, dk->type(), false, dk);
      else
        kvReduceBatchedForGQA<T, Context>(ctx, dk_tmp, dk);
    }

    if (dv) {
      if (dv->meta().is_contiguous())
        phi::SumKernel<T, Context>(ctx, dv_tmp, {3}, dv->type(), false, dv);
      else
        kvReduceBatchedForGQA<T, Context>(ctx, dv_tmp, dv);
    }
  }
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnGradKernel(const Context& ctx,
                         const DenseTensor& q,
                         const DenseTensor& k,
                         const DenseTensor& v,
                         const DenseTensor& out,
                         const DenseTensor& softmax_lse,
                         const DenseTensor& seed_offset,
                         const paddle::optional<DenseTensor>& attn_mask,
                         const DenseTensor& dout,
                         float dropout,
                         bool causal,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {
  if (dq) {
    ctx.template Alloc<T>(dq);
  }
  if (dk) {
    ctx.template Alloc<T>(dk);
  }
  if (dv) {
    ctx.template Alloc<T>(dv);
  }
  FlashAttnGradBaseKernel<T, Context>(ctx,
                                      q,
                                      k,
                                      v,
                                      out,
                                      softmax_lse,
                                      seed_offset,
                                      attn_mask,
                                      paddle::none,
                                      paddle::none,
                                      dout,
                                      dropout,
                                      causal,
                                      0,
                                      dq,
                                      dk,
                                      dv);
}

template <typename T, typename Context>
void FlashAttnQKVPackedGradKernel(
    const Context& ctx,
    const DenseTensor& qkv,
    const DenseTensor& out,
    const DenseTensor& softmax_lse,
    const DenseTensor& seed_offset,
    const paddle::optional<DenseTensor>& attn_mask,
    const DenseTensor& dout,
    float dropout,
    bool causal,
    DenseTensor* dqkv) {
#ifdef PADDLE_WITH_FLASHATTN
  // qkv [batchsize, seqlen, nheads/nheads_k+2, nheads_k, head_dim]
  const auto head_groupnum = qkv.dims()[2];  // nheads/nheads_k + 1 + 1
  DenseTensor q, k, v;
  sliceFlattenView(qkv, &q, 2, 0, head_groupnum - 2);
  sliceFlattenView(qkv, &k, 2, head_groupnum - 2, 1);
  sliceFlattenView(qkv, &v, 2, head_groupnum - 1, 1);
  // DenseTensor dqkv_tmp;
  if (!dqkv) {
    return;
    // dqkv is the only output. No need to compute if no dqkv
    // dqkv_tmp.Resize(qkv.dims());
    // dqkv = &dqkv_tmp;
  }
  ctx.template Alloc<T>(dqkv);
  DenseTensor dq, dk, dv;
  sliceFlattenView(*dqkv, &dq, 2, 0, head_groupnum - 2);
  sliceFlattenView(*dqkv, &dk, 2, head_groupnum - 2, 1);
  sliceFlattenView(*dqkv, &dv, 2, head_groupnum - 1, 1);
  FlashAttnGradBaseKernel<T, Context>(ctx,
                                      q,
                                      k,
                                      v,
                                      out,
                                      softmax_lse,
                                      seed_offset,
                                      attn_mask,
                                      paddle::none,
                                      paddle::none,
                                      dout,
                                      dropout,
                                      causal,
                                      0,
                                      &dq,
                                      &dk,
                                      &dv);
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnWithSparseGradKernel(
    const Context& ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const DenseTensor& attn_mask_start_row_indices,
    const DenseTensor& attn_mask_end_row_indices,
    const DenseTensor& out,
    const DenseTensor& softmax_lse,
    const DenseTensor& seed_offset,
    const DenseTensor& dout,
    float dropout,
    bool causal,
    int attn_mask_start_row,
    DenseTensor* dq,
    DenseTensor* dk,
    DenseTensor* dv) {
  if (dq) {
    ctx.template Alloc<T>(dq);
  }
  if (dk) {
    ctx.template Alloc<T>(dk);
  }
  if (dv) {
    ctx.template Alloc<T>(dv);
  }
  FlashAttnGradBaseKernel<T, Context>(ctx,
                                      q,
                                      k,
                                      v,
                                      out,
                                      softmax_lse,
                                      seed_offset,
                                      paddle::none,
                                      attn_mask_start_row_indices,
                                      attn_mask_end_row_indices,
                                      dout,
                                      dropout,
                                      causal,
                                      attn_mask_start_row,
                                      dq,
                                      dk,
                                      dv);
}
}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(7).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_varlen_qkvpacked_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnVarlenQKVPackedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_qkvpacked_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnQKVPackedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(3).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_with_sparse_mask_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnWithSparseGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(7).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}
