// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "paddle/phi/backends/dynload/hipblas.h"
#include "paddle/phi/backends/dynload/hipblaslt.h"
#include "paddle/phi/kernels/flash_attn_grad_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/complex_functors.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"

/*
#include <hipblas.h>
#include <hip/hip_runtime.h>
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11000
#include <hipblaslt.h>
#endif
*/

namespace phi {

template <typename T, typename Context>
void FlashAttnGradKernel(const Context& ctx,
                         const DenseTensor& q,
                         const DenseTensor& k,
                         const DenseTensor& v,
                         const DenseTensor& dout,
                         float dropout,
                         float scale,
                         bool causal,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
