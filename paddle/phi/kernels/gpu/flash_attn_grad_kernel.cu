#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/flash_attn_grad_kernel.h"
#include "glog/logging.h"  // For VLOG()
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/flags.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/arange_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/gpu/flash_attn_utils.h"
#include "paddle/phi/kernels/reshape_kernel.h"

PD_DECLARE_bool(cudnn_deterministic);

namespace phi {

int get_num_split() {
  // 0 for an internal heuristic, which is optimal
  return FLAGS_cudnn_deterministic ? 1 : 0;
}

template <typename T, typename Context>
void FlashAttnUnpaddedGradKernel(const Context& ctx,
                                 const DenseTensor& q,
                                 const DenseTensor& k,
                                 const DenseTensor& v,
                                 const DenseTensor& cu_seqlens_q,
                                 const DenseTensor& cu_seqlens_k,
                                 const DenseTensor& out,
                                 const DenseTensor& softmax_lse,
                                 const DenseTensor& seed_offset,
                                 const paddle::optional<DenseTensor>& attn_mask,
                                 const DenseTensor& dout,
                                 int64_t max_seqlen_q,
                                 int64_t max_seqlen_k,
                                 float scale,
                                 float dropout,
                                 bool causal,
                                 DenseTensor* dq,
                                 DenseTensor* dk,
                                 DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  ctx.template Alloc<T>(dq);
  ctx.template Alloc<T>(dk);
  ctx.template Alloc<T>(dv);

  const hipStream_t stream = ctx.stream();

  // q,k,v [total_*, num_heads, head_dim]
  auto dims = q.dims();

  const int64_t total_q = dims[0];
  const int64_t batch_size = cu_seqlens_q.numel() - 1;
  const int64_t num_heads = dims[1];
  const int64_t head_size_og = dout.dims()[2];
  const int64_t head_size = dims[2];
  const int64_t total_k = k.dims()[0];
  const int64_t num_heads_k = k.dims()[1];

  int num_splits = get_num_split();

  // TODO(umiswing): add shape check
  PADDLE_ENFORCE_EQ(
      head_size_og,
      head_size,
      phi::errors::InvalidArgument(
          "flash_attn_bwd receive input with head_size_og == head_size"));

  FlashAttnBwdParamsV2 params =
      FlashAttnBwdParamsV2(ctx,
                           batch_size,
                           max_seqlen_q,
                           max_seqlen_k,
                           num_heads,
                           num_heads_k,
                           head_size,
                           dropout,
                           scale,
                           causal,
                           q.dtype(),
                           attn_mask,
                           seed_offset.data<int64_t>());

  VLOG(10) << "FlashAttn bwd seed: " << params.seed
           << ", offset: " << params.offset;

  bool succ = phi::dynload::flash_attn_varlen_bwd(
      dout.data(),
      q.data(),
      k.data(),
      v.data(),
      out.data(),
      params.softmax_d.data(),
      softmax_lse.data(),
      cu_seqlens_q.data<int32_t>(),
      cu_seqlens_k.data<int32_t>(),
      params.rng_state.data(),
      dq->data(),
      dk->data(),
      dv->data(),
      params.dq_accum.data(),
      params.batch_size,
      params.max_seqlen_q,
      params.max_seqlen_k,
      params.seqlen_q_rounded,
      params.seqlen_k_rounded,
      params.num_heads,
      params.num_heads_k,
      params.head_size,
      params.head_size_rounded,
      params.dropout,
      params.scale,
      1.0f / params.scale,
      params.causal,
      params.is_bf16,
      num_splits,
      stream,
      params.seed,
      params.offset,
      params.attn_mask_tensor ? params.attn_mask_tensor->data() : nullptr,
      params.mask_dims.data());
  CheckFlashAttnStatus(succ);
#else
  RaiseNotSupportedError();
#endif
}

template <typename T, typename Context>
void FlashAttnGradKernel(const Context& ctx,
                         const DenseTensor& q,
                         const DenseTensor& k,
                         const DenseTensor& v,
                         const DenseTensor& out,
                         const DenseTensor& softmax_lse,
                         const DenseTensor& seed_offset,
                         const paddle::optional<DenseTensor>& attn_mask,
                         const DenseTensor& dout,
                         float dropout,
                         bool causal,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {
#ifdef PADDLE_WITH_FLASHATTN
  // q,k,v [batch_size, seq_len, num_heads, head_dim]

  auto dims = q.dims();
  int64_t batch_size = dims[0];
  int64_t seq_len_q = dims[1];
  int64_t num_heads = dims[2];
  int64_t head_size = dims[3];

  int64_t seq_len_k = k.dims()[1];

  int64_t total_q = batch_size * seq_len_q;
  int64_t total_k = batch_size * seq_len_k;

  float scale = 1.0f / std::sqrt(head_size);

  VLOG(4) << "FlashAttn bwd dims q[" << q.dims() << "], k[" << k.dims()
          << "], v[" << v.dims() << "]";

  DenseTensor q_t_s, k_t_s, v_t_s;
  q_t_s.ShareDataWith(q).Resize({total_q, num_heads, head_size});
  k_t_s.ShareDataWith(k).Resize({total_k, num_heads, head_size});
  v_t_s.ShareDataWith(v).Resize({total_k, num_heads, head_size});

  DenseTensor cu_seqlens_q;
  DenseTensor cu_seqlens_k;
  ArangeNullaryKernel<int32_t, Context>(
      ctx, 0, (batch_size + 1) * seq_len_q, seq_len_q, &cu_seqlens_q);
  ArangeNullaryKernel<int32_t, Context>(
      ctx, 0, (batch_size + 1) * seq_len_k, seq_len_k, &cu_seqlens_k);

  FlashAttnUnpaddedGradKernel<T, Context>(ctx,
                                          q_t_s,
                                          k_t_s,
                                          v_t_s,
                                          cu_seqlens_q,
                                          cu_seqlens_k,
                                          out,
                                          softmax_lse,
                                          seed_offset,
                                          dout,
                                          seq_len_q,
                                          seq_len_k,
                                          scale,
                                          dropout,
                                          causal,
                                          dq,
                                          dk,
                                          dv);

  VLOG(10) << "FlashAttn bwd seed: " << params.seed
           << ", offset: " << params.offset;

  int num_splits = get_num_split();

  bool succ = phi::dynload::flash_attn_bwd(
      dout.data(),
      q.data(),
      k.data(),
      v.data(),
      out.data(),
      params.softmax_d.data(),
      softmax_lse.data(),
      params.rng_state.data(),
      dq->data(),
      dk->data(),
      dv->data(),
      params.dq_accum.data(),
      params.batch_size,
      params.max_seqlen_q,
      params.max_seqlen_k,
      params.seqlen_q_rounded,
      params.seqlen_k_rounded,
      params.num_heads,
      params.num_heads_k,
      params.head_size,
      params.head_size_rounded,
      params.dropout,
      params.scale,
      std::sqrt(head_size),  // for unscale
      params.causal,
      params.is_bf16,
      num_splits,
      stream,
      params.seed,
      params.offset,
      params.attn_mask_tensor ? params.attn_mask_tensor->data() : nullptr,
      params.mask_dims.data());
  CheckFlashAttnStatus(succ);
#else
  RaiseNotSupportedError();
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(flash_attn_unpadded_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnUnpaddedGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(7).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}

PD_REGISTER_KERNEL(flash_attn_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::FlashAttnGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);  // seed_offset
}
