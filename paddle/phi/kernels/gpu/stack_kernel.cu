#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/stack_kernel.h"

#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T, typename IntType>
__global__ void StackCUDAKernel(T** input_ptrs,
                                IntType split_size,
                                IntType rows,
                                IntType cols,
                                T* __restrict__ output) {
  IntType grid_x = static_cast<IntType>(blockIdx.x) * blockDim.x + threadIdx.x;
  IntType grid_x_stride = static_cast<IntType>(blockDim.x) * gridDim.x;
  IntType grid_y_stride = static_cast<IntType>(blockDim.y) * gridDim.y;

  for (; grid_x < cols; grid_x += grid_x_stride) {
    IntType grid_y =
        static_cast<IntType>(blockIdx.y) * blockDim.y + threadIdx.y;

    IntType split = grid_x / split_size;
    const T* input_ptr = input_ptrs[split];
    IntType col_offset = grid_x % split_size;
#pragma unroll
    for (; grid_y < rows; grid_y += grid_y_stride) {
      output[grid_y * cols + grid_x] =
          input_ptr[grid_y * split_size + col_offset];
    }
  }
}

template <typename T, typename Context>
void StackKernel(const Context& dev_ctx,
                 const std::vector<const DenseTensor*>& x,
                 int axis,
                 DenseTensor* out) {
  if (axis < 0) axis += (x[0]->dims().size() + 1);

  int n = static_cast<int>(x.size());
  T* y_data = dev_ctx.template Alloc<T>(out);
  std::vector<const T*> x_datas(n);
  for (int i = 0; i < n; i++) {
    x_datas[i] = x[i]->data<T>();
  }

  auto tmp_x_data = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      x_datas.size() * sizeof(T*),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       tmp_x_data->ptr(),
                       phi::CPUPlace(),
                       reinterpret_cast<void*>(x_datas.data()),
                       x_datas.size() * sizeof(T*),
                       dev_ctx.stream());

  // Split x dim from axis to matrix
  int64_t x_row = 1, x_col = 1;
  for (int i = 0; i < axis; ++i) {
    x_row *= x[0]->dims()[i];
  }
  x_col = x[0]->numel() / x_row;
  int64_t out_col = x_col * n;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig2D(dev_ctx, out_col, x_row);

  if (out->numel() < std::numeric_limits<int32_t>::max()) {
    StackCUDAKernel<T, int32_t>
        <<<config.block_per_grid,
           config.thread_per_block,
           0,
           dev_ctx.stream()>>>(reinterpret_cast<T**>(tmp_x_data->ptr()),
                               static_cast<int32_t>(x_col),
                               static_cast<int32_t>(x_row),
                               static_cast<int32_t>(out_col),
                               y_data);
  } else {
    StackCUDAKernel<T, int64_t>
        <<<config.block_per_grid,
           config.thread_per_block,
           0,
           dev_ctx.stream()>>>(reinterpret_cast<T**>(tmp_x_data->ptr()),
                               x_col,
                               x_row,
                               out_col,
                               y_data);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(stack,
                   GPU,
                   ALL_LAYOUT,
                   phi::StackKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
