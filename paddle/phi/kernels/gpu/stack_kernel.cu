#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/stack_kernel.h"
<<<<<<< HEAD
#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/segmented_array.h"

namespace phi {

template <typename T, typename IndexT, typename ArrayT>
__global__ void StackCUDAKernel(ArrayT array,
                                funcs::GeneralDivMod<IndexT> divmoder,
                                IndexT split_size,
                                IndexT rows,
                                IndexT cols,
                                T* __restrict__ output) {
  IndexT grid_x = static_cast<IndexT>(blockIdx.x) * blockDim.x + threadIdx.x;
  IndexT grid_x_stride = static_cast<IndexT>(blockDim.x) * gridDim.x;
  IndexT grid_y_stride = static_cast<IndexT>(blockDim.y) * gridDim.y;

  for (; grid_x < cols; grid_x += grid_x_stride) {
    IndexT grid_y = static_cast<IndexT>(blockIdx.y) * blockDim.y + threadIdx.y;

    auto divmod_rslt = divmoder.div_mod(grid_x);
    IndexT split = divmod_rslt[0];       // grid_x / split_size
    IndexT col_offset = divmod_rslt[1];  // grid_x % split_size
    const T* input_ptr = array.data[split];
=======

#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T, typename IntType>
__global__ void StackCUDAKernel(T** input_ptrs,
                                IntType split_size,
                                IntType rows,
                                IntType cols,
                                T* __restrict__ output) {
  IntType grid_x = static_cast<IntType>(blockIdx.x) * blockDim.x + threadIdx.x;
  IntType grid_x_stride = static_cast<IntType>(blockDim.x) * gridDim.x;
  IntType grid_y_stride = static_cast<IntType>(blockDim.y) * gridDim.y;

  for (; grid_x < cols; grid_x += grid_x_stride) {
    IntType grid_y =
        static_cast<IntType>(blockIdx.y) * blockDim.y + threadIdx.y;

    IntType split = grid_x / split_size;
    const T* input_ptr = input_ptrs[split];
    IntType col_offset = grid_x % split_size;
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
#pragma unroll
    for (; grid_y < rows; grid_y += grid_y_stride) {
      output[grid_y * cols + grid_x] =
          input_ptr[grid_y * split_size + col_offset];
    }
  }
}

<<<<<<< HEAD
template <typename Context,
          typename T,
          typename IndexT,
          funcs::SegmentedArraySize Size>
void LaunchStackKernel(const Context& ctx,
                       const IndexT x_col,
                       const IndexT x_row,
                       const IndexT out_col,
                       const std::vector<const DenseTensor*>& x,
                       DenseTensor* out) {
  T* out_ptr = ctx.template Alloc<T>(out);
  auto config = phi::backends::gpu::GetGpuLaunchConfig2D(ctx, out_col, x_row);

  funcs::ConstPointerArraySetter<Context, T, Size> setter(ctx, x);
  funcs::GeneralDivMod<IndexT> divmoder(x_col);
  StackCUDAKernel<T, IndexT, decltype(setter.array)>
      <<<config.block_per_grid, config.thread_per_block, 0, ctx.stream()>>>(
          setter.array, divmoder, x_col, x_row, out_col, out_ptr);
}

template <typename T, typename Context>
void StackKernel(const Context& ctx,
=======
template <typename T, typename Context>
void StackKernel(const Context& dev_ctx,
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
                 const std::vector<const DenseTensor*>& x,
                 int axis,
                 DenseTensor* out) {
  if (axis < 0) axis += (x[0]->dims().size() + 1);
<<<<<<< HEAD
  int num = static_cast<int>(x.size());

  // Split x dim from axis to matrix
  int64_t x_row = 1;
  for (int i = 0; i < axis; ++i) {
    x_row *= x[0]->dims()[i];
  }
  int64_t x_col = x[0]->numel() / x_row;
  int64_t out_col = x_col * num;

  if (out->numel() < std::numeric_limits<int32_t>::max()) {
    switch (funcs::CalcArraySize(num)) {
      SEGMENTED_ARRAY_KERNEL_HELPER(
          LaunchStackKernel<Context, T, int32_t, kArraySize>(
              ctx, x_col, x_row, out_col, x, out));
    }
  } else {
    switch (funcs::CalcArraySize(num)) {
      SEGMENTED_ARRAY_KERNEL_HELPER(
          LaunchStackKernel<Context, T, int64_t, kArraySize>(
              ctx, x_col, x_row, out_col, x, out));
    }
=======

  int n = static_cast<int>(x.size());
  T* y_data = dev_ctx.template Alloc<T>(out);
  std::vector<const T*> x_datas(n);
  for (int i = 0; i < n; i++) {
    x_datas[i] = x[i]->data<T>();
  }

  auto tmp_x_data = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      x_datas.size() * sizeof(T*),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       tmp_x_data->ptr(),
                       phi::CPUPlace(),
                       reinterpret_cast<void*>(x_datas.data()),
                       x_datas.size() * sizeof(T*),
                       dev_ctx.stream());

  // Split x dim from axis to matrix
  int64_t x_row = 1, x_col = 1;
  for (int i = 0; i < axis; ++i) {
    x_row *= x[0]->dims()[i];
  }
  x_col = x[0]->numel() / x_row;
  int64_t out_col = x_col * n;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig2D(dev_ctx, out_col, x_row);

  if (out->numel() < std::numeric_limits<int32_t>::max()) {
    StackCUDAKernel<T, int32_t>
        <<<config.block_per_grid,
           config.thread_per_block,
           0,
           dev_ctx.stream()>>>(reinterpret_cast<T**>(tmp_x_data->ptr()),
                               static_cast<int32_t>(x_col),
                               static_cast<int32_t>(x_row),
                               static_cast<int32_t>(out_col),
                               y_data);
  } else {
    StackCUDAKernel<T, int64_t>
        <<<config.block_per_grid,
           config.thread_per_block,
           0,
           dev_ctx.stream()>>>(reinterpret_cast<T**>(tmp_x_data->ptr()),
                               x_col,
                               x_row,
                               out_col,
                               y_data);
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(stack,
                   GPU,
                   ALL_LAYOUT,
                   phi::StackKernel,
                   float,
                   double,
<<<<<<< HEAD
                   bool,
                   int64_t,
                   int,
                   uint8_t,
                   int8_t,
=======
                   int64_t,
                   int,
>>>>>>> 0699afb112355f7e0a08b05030bb7fe613554d81
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
