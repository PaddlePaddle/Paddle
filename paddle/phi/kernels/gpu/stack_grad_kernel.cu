#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/stack_grad_kernel.h"

#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T, typename IntType>
__global__ void UnStackHelperCUDAKernel(const T* __restrict__ input,
                                        int pre_dim_size,
                                        int split_dim_size,
                                        int suf_dim_size,
                                        int num_split,
                                        T** output_ptrs) {
  assert(blockDim.y == 1);
  assert(blockDim.z == 1);
  // In this case they are equal
  assert(split_dim_size % num_split == 0);

  IntType size = pre_dim_size * split_dim_size * suf_dim_size;
  IntType each_dim_size = split_dim_size / num_split;

  for (IntType offset = blockIdx.x * blockDim.x + threadIdx.x; offset < size;
       offset += blockDim.x * gridDim.x) {
    IntType i = offset / (split_dim_size * suf_dim_size);
    IntType j = (offset % (split_dim_size * suf_dim_size)) / suf_dim_size;
    IntType k = offset % suf_dim_size;

    T* output = output_ptrs[j / each_dim_size];
    if (output == nullptr) {
      return;
    }
    IntType output_ind = i * each_dim_size * suf_dim_size +
                         (j % each_dim_size) * suf_dim_size + k;
    *(output + output_ind) = input[offset];
  }
}

template <typename T, typename Context>
void StackGradKernel(const Context& dev_ctx,
                     const DenseTensor& out,
                     int axis,
                     std::vector<DenseTensor*> x_grad) {
  if (axis < 0) axis += out.dims().size();

  int n = out.dims()[axis];
  PADDLE_ENFORCE_EQ(n,
                    x_grad.size(),
                    phi::errors::InvalidArgument(
                        "Output x_grad size should be equal to n, but"
                        " received n is:%d x_grad size is:%d.",
                        n,
                        x_grad.size()));

  // x_grad is output, so save each data address, then copy each dy into dx_data
  std::vector<T*> outputs(n);
  for (size_t j = 0; j < x_grad.size(); ++j) {
    if (x_grad[j] == nullptr) {
      outputs[j] = nullptr;
      continue;
    }
    if (x_grad[j]->numel() != 0UL) {
      T* ptr = dev_ctx.template Alloc<T>(x_grad[j]);
      outputs[j] = ptr;
    } else {
      outputs[j] = nullptr;
    }
  }
  auto dy_data = out.data<T>();
  // each x_grad should have same shape
  int dy_pre = 1, dy_suf = 1;
  auto dy_dims = out.dims();
  int split_dim = n;
  for (int i = 0; i < axis; ++i) {
    dy_pre *= dy_dims[i];
  }
  dy_suf = out.numel() / (split_dim * dy_pre);

  auto tmp_out_data = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      outputs.size() * sizeof(T*),
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  paddle::memory::Copy(dev_ctx.GetPlace(),
                       tmp_out_data->ptr(),
                       phi::CPUPlace(),
                       reinterpret_cast<void*>(outputs.data()),
                       outputs.size() * sizeof(T*),
                       dev_ctx.stream());

  auto config = phi::backends::gpu::GetGpuLaunchConfig1D(
      dev_ctx, dy_pre * split_dim * dy_suf);

  if (out.numel() < std::numeric_limits<int32_t>::max()) {
    UnStackHelperCUDAKernel<T, int32_t>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(dy_data,
                               dy_pre,
                               split_dim,
                               dy_suf,
                               split_dim,
                               reinterpret_cast<T**>(tmp_out_data->ptr()));
  } else {
    UnStackHelperCUDAKernel<T, int64_t>
        <<<config.block_per_grid.x,
           config.thread_per_block.x,
           0,
           dev_ctx.stream()>>>(dy_data,
                               dy_pre,
                               split_dim,
                               dy_suf,
                               split_dim,
                               reinterpret_cast<T**>(tmp_out_data->ptr()));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(stack_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::StackGradKernel,
                   float,
                   double,
                   int64_t,
                   int,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
