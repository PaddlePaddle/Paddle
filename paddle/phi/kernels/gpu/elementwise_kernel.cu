#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/complex.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/impl/elementwise_kernel_impl.h"

namespace phi {

#define DEFINE_CUDA_ELEMENTWISE_OP(name)                             \
  template <typename T, typename Context>                            \
  void name##RawKernel(const Context& dev_ctx,                       \
                       const DenseTensor& x,                         \
                       const DenseTensor& y,                         \
                       int axis,                                     \
                       DenseTensor* out) {                           \
    std::vector<const DenseTensor*> inputs;                          \
    std::vector<DenseTensor*> outputs;                               \
    inputs.emplace_back(&x);                                         \
    inputs.emplace_back(&y);                                         \
    outputs.emplace_back(out);                                       \
    dev_ctx.template Alloc<T>(out);                                  \
    funcs::BroadcastKernel<ElementwiseType::kBinary, T, T>(          \
        dev_ctx, inputs, &outputs, axis, funcs::name##Functor<T>()); \
  }

/**
 * Kernels
 */

// Create the definition of Add
DEFINE_CUDA_ELEMENTWISE_OP(Add)
// Create the definition of Subtract
DEFINE_CUDA_ELEMENTWISE_OP(Subtract)
// Create the definition of Multiply
DEFINE_CUDA_ELEMENTWISE_OP(Multiply)
// Create the definition of Divide
DEFINE_CUDA_ELEMENTWISE_OP(Divide)
// Create the definition of Maximum
DEFINE_CUDA_ELEMENTWISE_OP(Maximum)
// Create the definition of Minimum
DEFINE_CUDA_ELEMENTWISE_OP(Minimum)
// Create the definition of Modulo
DEFINE_CUDA_ELEMENTWISE_OP(Modulo)
// Create the definition of FloorDivide
DEFINE_CUDA_ELEMENTWISE_OP(FloorDivide)
// Create the definition of Pow
DEFINE_CUDA_ELEMENTWISE_OP(ElementwisePow)

}  // namespace phi

using float16 = phi::dtype::float16;
using bfloat16 = phi::dtype::bfloat16;
using complex64 = ::phi::dtype::complex<float>;
using complex128 = ::phi::dtype::complex<double>;

PD_REGISTER_KERNEL(
    fmax, GPU, ALL_LAYOUT, phi::FMaxKernel, float, double, int, int64_t) {}

PD_REGISTER_KERNEL(
    fmin, GPU, ALL_LAYOUT, phi::FMinKernel, float, double, int, int64_t) {}

PD_REGISTER_KERNEL(add_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::AddRawKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   int64_t,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}
PD_REGISTER_KERNEL(subtract_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::SubtractRawKernel,
                   float,
                   double,
                   int16_t,
                   int,
                   int64_t,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}
PD_REGISTER_KERNEL(divide_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::DivideRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   bfloat16,
                   complex64,
                   complex128) {}
PD_REGISTER_KERNEL(multiply_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::MultiplyRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   bool,
                   float16,
                   complex64,
                   complex128,
                   bfloat16) {}
PD_REGISTER_KERNEL(maximum_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaximumRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   bfloat16) {}
PD_REGISTER_KERNEL(minimum_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::MinimumRawKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   float16,
                   bfloat16) {}
PD_REGISTER_KERNEL(modulo_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::ModuloRawKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
PD_REGISTER_KERNEL(floor_divide_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::FloorDivideRawKernel,
                   int,
                   int64_t) {}
PD_REGISTER_KERNEL(elementwise_pow_raw,
                   GPU,
                   ALL_LAYOUT,
                   phi::ElementwisePowRawKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
