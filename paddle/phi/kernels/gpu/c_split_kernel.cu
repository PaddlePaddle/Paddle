#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/c_split_kernel.h"

#include "paddle/phi/core/kernel_registry.h"

namespace phi {

static constexpr int64_t kNumCUDAThreads = 512;
static constexpr int64_t kNumMaxinumNumBlocks = 4096;

static inline int64_t NumBlocks(const int64_t N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T>
__global__ void SplitFromRank(const T* input,
                              T* output,
                              const int64_t rows,
                              const int64_t columns,
                              const int rank,
                              const int nranks,
                              const int64_t limit) {
  CUDA_KERNEL_LOOP_TYPE(i, limit, int64_t) {
    int64_t row = i / columns;
    int64_t col = i % columns;

    int64_t block = columns / nranks;
    int64_t start = block * rank;
    int64_t end = start + block;

    if (col >= start && col < end) {
      int64_t idx = block * row + col % block;
      output[idx] = input[i];
    }
  }
}

template <typename T, typename Context>
void CSplitKernel(const Context& ctx,
                  const DenseTensor& x,
                  int rank,
                  int nranks,
                  int ring_id,
                  bool use_calc_stream,
                  bool use_model_parallel,
                  DenseTensor* out) {
  auto place = ctx.GetPlace();

  PADDLE_ENFORCE_GE(rank,
                    0,
                    common::errors::PreconditionNotMet(
                        "The value of rank (%d) for c_split must be "
                        "greater than or equal to 0.",
                        rank));
  PADDLE_ENFORCE_GE(nranks,
                    2,
                    common::errors::PreconditionNotMet(
                        "The value of nranks (%d) for c_split must be "
                        "greater than or equal to 2.",
                        nranks));
  PADDLE_ENFORCE_LT(rank,
                    nranks,
                    common::errors::PreconditionNotMet(
                        "The value of rank (%d) for c_split must be "
                        "less than that of nranks (%d).",
                        rank,
                        nranks));

  auto dims = x.dims();
  auto dims_size = dims.size();
  // final dim
  int64_t end_size = dims[dims_size - 1];

  // remain dim
  auto remain_ddim = common::slice_ddim(dims, 0, dims_size - 1);
  int64_t remain_numel = common::product(remain_ddim);

  int64_t limit = x.numel();
  int64_t blocks = NumBlocks(limit);
  int64_t threads = kNumCUDAThreads;

  dims[dims_size - 1] /= nranks;
  out->Resize(dims);
  ctx.template Alloc<T>(out);

  SplitFromRank<T><<<blocks, threads, 0, ctx.stream()>>>(
      x.data<T>(), out->data<T>(), remain_numel, end_size, rank, nranks, limit);
}

}  // namespace phi

#if (NCCL_VERSION_CODE >= 21000 && CUDA_VERSION >= 11000) || \
    defined(PADDLE_WITH_HIP)
PD_REGISTER_KERNEL(c_split,
                   GPU,
                   ALL_LAYOUT,
                   phi::CSplitKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::bfloat16,
                   phi::dtype::float16) {}
#else
PD_REGISTER_KERNEL(c_split,
                   GPU,
                   ALL_LAYOUT,
                   phi::CSplitKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
#endif
