#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/pad3d_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename T>
__global__ void Pad3DGradConstNCDHW(const int in_size,
                                    T* d_in_data,
                                    const int num,
                                    const int channels,
                                    const int in_depth,
                                    const int in_height,
                                    const int in_width,
                                    const int out_depth,
                                    const int out_height,
                                    const int out_width,
                                    const int pad_front,
                                    const int pad_top,
                                    const int pad_left,
                                    const T* d_out_data) {
  CUDA_KERNEL_LOOP(in_index, in_size) {
    const int in_w = in_index % in_width;

    int nc = in_index / in_width;
    const int in_h = nc % in_height;

    nc /= in_height;
    const int in_d = nc % in_depth;

    nc /= in_depth;

    const int out_d = in_d + pad_front;
    const int out_h = in_h + pad_top;
    const int out_w = in_w + pad_left;
    d_in_data[in_index] =
        d_out_data[nc * out_depth * out_height * out_width +
                   out_d * out_height * out_width + out_h * out_width + out_w];
  }
}

template <typename T>
__global__ void Pad3DGradConstNDHWC(const int in_size,
                                    T* d_in_data,
                                    const int num,
                                    const int channels,
                                    const int in_depth,
                                    const int in_height,
                                    const int in_width,
                                    const int out_depth,
                                    const int out_height,
                                    const int out_width,
                                    const int pad_front,
                                    const int pad_top,
                                    const int pad_left,
                                    const T* d_out_data) {
  CUDA_KERNEL_LOOP(in_index, in_size) {
    const int c = in_index % channels;
    int n = in_index / channels;

    const int in_w = n % in_width;
    n /= in_width;

    const int in_h = n % in_height;
    n /= in_height;

    const int in_d = n % in_depth;
    n /= in_depth;

    const int out_d = in_d + pad_front;
    const int out_h = in_h + pad_top;
    const int out_w = in_w + pad_left;

    d_in_data[in_index] =
        d_out_data[n * out_depth * out_height * out_width * channels +
                   out_d * out_height * out_width * channels +
                   out_h * out_width * channels + out_w * channels + c];
  }
}

template <typename T>
__global__ void Pad3DGradReflectNCDHW(const int out_size,
                                      T* d_in_data,
                                      const int num,
                                      const int channels,
                                      const int in_depth,
                                      const int in_height,
                                      const int in_width,
                                      const int out_depth,
                                      const int out_height,
                                      const int out_width,
                                      const int pad_front,
                                      const int pad_top,
                                      const int pad_left,
                                      const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    int nc = out_index / out_width;
    const int out_w = out_index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);
    in_h = max(in_h, -in_h);
    in_w = max(in_w, -in_w);

    in_d = min(in_d, 2 * in_depth - in_d - 2);
    in_h = min(in_h, 2 * in_height - in_h - 2);
    in_w = min(in_w, 2 * in_width - in_w - 2);

    paddle::platform::CudaAtomicAdd(
        &d_in_data[nc * in_depth * in_height * in_width +
                   in_d * in_height * in_width + in_h * in_width + in_w],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradReflectNDHWC(const int out_size,
                                      T* d_in_data,
                                      const int num,
                                      const int channels,
                                      const int in_depth,
                                      const int in_height,
                                      const int in_width,
                                      const int out_depth,
                                      const int out_height,
                                      const int out_width,
                                      const int pad_front,
                                      const int pad_top,
                                      const int pad_left,
                                      const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    const int c = out_index % channels;
    int n = out_index / channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = out_d - pad_front;
    int in_h = out_h - pad_top;
    int in_w = out_w - pad_left;

    in_d = max(in_d, -in_d);
    in_h = max(in_h, -in_h);
    in_w = max(in_w, -in_w);

    in_d = min(in_d, in_depth * 2 - in_d - 2);
    in_h = min(in_h, in_height * 2 - in_h - 2);
    in_w = min(in_w, in_width * 2 - in_w - 2);
    paddle::platform::CudaAtomicAdd(
        &d_in_data[n * in_depth * in_height * in_width * channels +
                   in_d * in_height * in_width * channels +
                   in_h * in_width * channels + in_w * channels + c],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradReplicateNCDHW(const int out_size,
                                        T* d_in_data,
                                        const int num,
                                        const int channels,
                                        const int in_depth,
                                        const int in_height,
                                        const int in_width,
                                        const int out_depth,
                                        const int out_height,
                                        const int out_width,
                                        const int pad_front,
                                        const int pad_top,
                                        const int pad_left,
                                        const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    int nc = out_index / out_width;
    const int out_w = out_index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    const int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    const int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    const int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    paddle::platform::CudaAtomicAdd(
        &d_in_data[nc * in_depth * in_height * in_width +
                   in_d * in_height * in_width + in_h * in_width + in_w],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradReplicateNDHWC(const int out_size,
                                        T* d_in_data,
                                        const int num,
                                        const int channels,
                                        const int in_depth,
                                        const int in_height,
                                        const int in_width,
                                        const int out_depth,
                                        const int out_height,
                                        const int out_width,
                                        const int pad_front,
                                        const int pad_top,
                                        const int pad_left,
                                        const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    const int c = out_index % channels;
    int n = out_index / channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    const int in_d = min(in_depth - 1, max(out_d - pad_front, 0));
    const int in_h = min(in_height - 1, max(out_h - pad_top, 0));
    const int in_w = min(in_width - 1, max(out_w - pad_left, 0));

    paddle::platform::CudaAtomicAdd(
        &d_in_data[n * in_depth * in_height * in_width * channels +
                   in_d * in_height * in_width * channels +
                   in_h * in_width * channels + in_w * channels + c],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradCircularNCDHW(const int out_size,
                                       T* d_in_data,
                                       const int num,
                                       const int channels,
                                       const int in_depth,
                                       const int in_height,
                                       const int in_width,
                                       const int out_depth,
                                       const int out_height,
                                       const int out_width,
                                       const int pad_front,
                                       const int pad_top,
                                       const int pad_left,
                                       const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    int nc = out_index / out_width;
    const int out_w = out_index % out_width;
    const int out_h = nc % out_height;
    nc /= out_height;
    const int out_d = nc % out_depth;
    nc /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    paddle::platform::CudaAtomicAdd(
        &d_in_data[nc * in_depth * in_height * in_width +
                   in_d * in_height * in_width + in_h * in_width + in_w],
        d_out_data[out_index]);
  }
}

template <typename T>
__global__ void Pad3DGradCircularNDHWC(const int out_size,
                                       T* d_in_data,
                                       const int num,
                                       const int channels,
                                       const int in_depth,
                                       const int in_height,
                                       const int in_width,
                                       const int out_depth,
                                       const int out_height,
                                       const int out_width,
                                       const int pad_front,
                                       const int pad_top,
                                       const int pad_left,
                                       const T* d_out_data) {
  CUDA_KERNEL_LOOP(out_index, out_size) {
    const int c = out_index % channels;
    int n = out_index / channels;
    const int out_w = n % out_width;
    n /= out_width;
    const int out_h = n % out_height;
    n /= out_height;
    const int out_d = n % out_depth;
    n /= out_depth;

    int in_d = ((out_d - pad_front) % in_depth + in_depth) % in_depth;
    int in_h = ((out_h - pad_top) % in_height + in_height) % in_height;
    int in_w = ((out_w - pad_left) % in_width + in_width) % in_width;

    paddle::platform::CudaAtomicAdd(
        &d_in_data[n * in_depth * in_height * in_width * channels +
                   in_d * in_height * in_width * channels +
                   in_h * in_width * channels + in_w * channels + c],
        d_out_data[out_index]);
  }
}

template <typename T, typename Context>
void Pad3dGradKernel(const Context& dev_ctx,
                     const DenseTensor& x,
                     const DenseTensor& out_grad,
                     const IntArray& paddings,
                     const std::string& mode,
                     float pad_value,
                     const std::string& data_format,
                     DenseTensor* x_grad) {
  std::vector<int64_t> pads = paddings.GetData();
  auto* d_out = &out_grad;
  auto* d_in = x_grad;
  auto d_in_dims = d_in->dims();
  auto d_out_dims = d_out->dims();
  const T* d_out_data = d_out->data<T>();
  T* d_in_data = dev_ctx.template Alloc<T>(d_in);

  phi::funcs::SetConstant<Context, T>()(dev_ctx, d_in, static_cast<T>(0));

  const int pad_left = pads[0];
  const int pad_top = pads[2];
  const int pad_front = pads[4];

  const int num = d_in_dims[0];

  auto stream = dev_ctx.stream();
  int block = PADDLE_CUDA_NUM_THREADS;
  const int out_size = d_out->numel();
  const int in_size = d_in->numel();
  int grid = (out_size + block - 1) / block;

  if (data_format == "NCDHW") {
    const int channels = d_in_dims[1];
    const int in_depth = d_in_dims[2];
    const int in_height = d_in_dims[3];
    const int in_width = d_in_dims[4];
    const int out_depth = d_out_dims[2];
    const int out_height = d_out_dims[3];
    const int out_width = d_out_dims[4];

    if (mode == "reflect") {
      Pad3DGradReflectNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                           d_in_data,
                                                           num,
                                                           channels,
                                                           in_depth,
                                                           in_height,
                                                           in_width,
                                                           out_depth,
                                                           out_height,
                                                           out_width,
                                                           pad_front,
                                                           pad_top,
                                                           pad_left,
                                                           d_out_data);
    } else if (mode == "replicate") {
      Pad3DGradReplicateNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                             d_in_data,
                                                             num,
                                                             channels,
                                                             in_depth,
                                                             in_height,
                                                             in_width,
                                                             out_depth,
                                                             out_height,
                                                             out_width,
                                                             pad_front,
                                                             pad_top,
                                                             pad_left,
                                                             d_out_data);
    } else if (mode == "circular") {
      Pad3DGradCircularNCDHW<T><<<grid, block, 0, stream>>>(out_size,
                                                            d_in_data,
                                                            num,
                                                            channels,
                                                            in_depth,
                                                            in_height,
                                                            in_width,
                                                            out_depth,
                                                            out_height,
                                                            out_width,
                                                            pad_front,
                                                            pad_top,
                                                            pad_left,
                                                            d_out_data);
    } else {
      grid = (in_size + block - 1) / block;
      Pad3DGradConstNCDHW<T><<<grid, block, 0, stream>>>(in_size,
                                                         d_in_data,
                                                         num,
                                                         channels,
                                                         in_depth,
                                                         in_height,
                                                         in_width,
                                                         out_depth,
                                                         out_height,
                                                         out_width,
                                                         pad_front,
                                                         pad_top,
                                                         pad_left,
                                                         d_out_data);
    }
  } else {
    const int channels = d_in_dims[4];
    const int in_depth = d_in_dims[1];
    const int in_height = d_in_dims[2];
    const int in_width = d_in_dims[3];
    const int out_depth = d_out_dims[1];
    const int out_height = d_out_dims[2];
    const int out_width = d_out_dims[3];
    if (mode == "reflect") {
      Pad3DGradReflectNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                           d_in_data,
                                                           num,
                                                           channels,
                                                           in_depth,
                                                           in_height,
                                                           in_width,
                                                           out_depth,
                                                           out_height,
                                                           out_width,
                                                           pad_front,
                                                           pad_top,
                                                           pad_left,
                                                           d_out_data);
    } else if (mode == "replicate") {
      Pad3DGradReplicateNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                             d_in_data,
                                                             num,
                                                             channels,
                                                             in_depth,
                                                             in_height,
                                                             in_width,
                                                             out_depth,
                                                             out_height,
                                                             out_width,
                                                             pad_front,
                                                             pad_top,
                                                             pad_left,
                                                             d_out_data);
    } else if (mode == "circular") {
      Pad3DGradCircularNDHWC<T><<<grid, block, 0, stream>>>(out_size,
                                                            d_in_data,
                                                            num,
                                                            channels,
                                                            in_depth,
                                                            in_height,
                                                            in_width,
                                                            out_depth,
                                                            out_height,
                                                            out_width,
                                                            pad_front,
                                                            pad_top,
                                                            pad_left,
                                                            d_out_data);
    } else {
      grid = (in_size + block - 1) / block;
      Pad3DGradConstNDHWC<T><<<grid, block, 0, stream>>>(in_size,
                                                         d_in_data,
                                                         num,
                                                         channels,
                                                         in_depth,
                                                         in_height,
                                                         in_width,
                                                         out_depth,
                                                         out_height,
                                                         out_width,
                                                         pad_front,
                                                         pad_top,
                                                         pad_left,
                                                         d_out_data);
    }
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(pad3d_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::Pad3dGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
