// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/kthvalue_kernel.h"

#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/eigen/eigen_function.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {
inline int getBlockSize(int col) {
  if (col > 512)
    return 1024;
  else if (col > 256 && col <= 512)
    return 512;
  else if (col > 128 && col <= 256)
    return 256;
  else if (col > 64 && col <= 128)
    return 128;
  else
    return 64;
}

template <typename T>
bool SortKthvalue(const phi::GPUContext& dev_ctx,
                  const DenseTensor* input_tensor,
                  const int64_t num_cols,
                  const int64_t num_rows,
                  const int k,
                  DenseTensor* out_tensor,
                  DenseTensor* indices_tensor) {
  auto cu_stream = dev_ctx.stream();
  DenseTensor input_indices;
  const std::vector<int64_t> dims = {num_rows, num_cols};
  auto dim = phi::make_ddim(dims);
  input_indices.Resize(dim);
  dev_ctx.template Alloc<int64_t>(&input_indices);
  size_t temp_storage_bytes = -1;
  int block_size = getBlockSize(num_cols);
  unsigned int maxGridDimX = dev_ctx.GetCUDAMaxGridDimSize()[0];
  unsigned int grid_size = num_rows < maxGridDimX
                               ? static_cast<unsigned int>(num_rows)
                               : maxGridDimX;
  paddle::operators::InitIndex<int64_t>
      <<<grid_size, block_size, 0, cu_stream>>>(
          input_indices.data<int64_t>(), num_rows, num_cols);
  hipcub::CountingInputIterator<int64_t> counting_iter(0);
  hipcub::TransformInputIterator<int64_t,
                              paddle::operators::SegmentOffsetIter,
                              hipcub::CountingInputIterator<int64_t>>
      segment_offsets_t(counting_iter,
                        paddle::operators::SegmentOffsetIter(num_cols));
  T* sorted_values_ptr;
  int64_t* sorted_indices_ptr;
  DenseTensor temp_values, temp_indices;
  const T* input = input_tensor->data<T>();
  T* values = out_tensor->data<T>();
  int64_t* indices = indices_tensor->mutable_data<int64_t>(dev_ctx.GetPlace());
  temp_values.Resize(dim);
  temp_indices.Resize(dim);
  sorted_values_ptr = dev_ctx.template Alloc<T>(&temp_values);
  sorted_indices_ptr = dev_ctx.template Alloc<int64_t>(&temp_indices);
  auto err =
      hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr,
                                               temp_storage_bytes,
                                               input,
                                               sorted_values_ptr,
                                               input_indices.data<int64_t>(),
                                               sorted_indices_ptr,
                                               num_cols * num_rows,
                                               num_rows,
                                               segment_offsets_t,
                                               segment_offsets_t + 1,
                                               0,
                                               sizeof(T) * 8,
                                               cu_stream);
#ifdef __HIPCC__
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, status: "
               << hipGetErrorString(err);
    return false;
  }
#else
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, status: "
               << hipGetErrorString(err);
    return false;
  }
#endif
  DenseTensor temp_storage;
  temp_storage.Resize({static_cast<int>(temp_storage_bytes / sizeof(uint8_t))});
  uint8_t* temp_storage_data = dev_ctx.template Alloc<uint8_t>(&temp_storage);

  err = hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage_data,
                                                 temp_storage_bytes,
                                                 input,
                                                 sorted_values_ptr,
                                                 input_indices.data<int64_t>(),
                                                 sorted_indices_ptr,
                                                 num_cols * num_rows,
                                                 num_rows,
                                                 segment_offsets_t,
                                                 segment_offsets_t + 1,
                                                 0,
                                                 sizeof(T) * 8,
                                                 cu_stream);
#ifdef __HIPCC__
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, "
               << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
#else
  if (err != hipSuccess) {
    LOG(ERROR) << "KthvalueOP failed as could not launch "
                  "hipcub::DeviceSegmentedRadixSort::SortPairs, "
               << temp_storage_bytes << ", status: " << hipGetErrorString(err);
    return false;
  }
#endif
  auto& dev = *dev_ctx.eigen_device();
  const Eigen::DSizes<Eigen::DenseIndex, 2> slice_indices{0, k - 1};
  const Eigen::DSizes<Eigen::DenseIndex, 2> slice_sizes{num_rows, 1};
  auto e_indices = EigenMatrix<int64_t>::From(*indices_tensor, dim);
  auto e_tmp_indices =
      EigenMatrix<int64_t>::From(static_cast<const DenseTensor>(temp_indices));
  std::vector<int> odims = {static_cast<int>(num_rows), static_cast<int>(1)};
  dim = phi::make_ddim(odims);
  auto e_values = EigenMatrix<T>::From(*out_tensor, dim);
  auto e_tmp_values =
      EigenMatrix<T>::From(static_cast<const DenseTensor>(temp_values));

  funcs::EigenSlice<std::decay_t<decltype(dev)>, int64_t, 2>::Eval(
      dev, e_indices, e_tmp_indices, slice_indices, slice_sizes);
  funcs::EigenSlice<std::decay_t<decltype(dev)>, T, 2>::Eval(
      dev, e_values, e_tmp_values, slice_indices, slice_sizes);
  return true;
}

template <typename T, typename Context>
void KthvalueKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    int k,
                    int axis,
                    bool keepdim,
                    DenseTensor* output,
                    DenseTensor* indices) {
  const auto& in_dims = x.dims();
  if (axis < 0) axis += in_dims.size();
  auto out_dims = output->dims();
  const T* input_data = x.data<T>();
  T* output_data = dev_ctx.template Alloc<T>(output);
  int64_t* indices_data = dev_ctx.template Alloc<int64_t>(indices);

  if (axis == in_dims.size() - 1) {
    const int64_t& input_height =
        phi::product(phi::slice_ddim(in_dims, 0, in_dims.size() - 1));
    const int64_t& input_width = in_dims[in_dims.size() - 1];
    PADDLE_ENFORCE_EQ(
        SortKthvalue<T>(
            dev_ctx, &x, input_width, input_height, k, output, indices),
        true,
        phi::errors::External("KthvalueOP: Error when use cub sorting"));
    return;
  } else {
    std::vector<int> trans;
    for (int i = 0; i < axis; i++) {
      trans.emplace_back(i);
    }
    trans.emplace_back(in_dims.size() - 1);
    for (int i = axis + 1; i < in_dims.size() - 1; i++) {
      trans.emplace_back(i);
    }
    trans.emplace_back(axis);
    if (!keepdim) {
      std::vector<int> tmp_out_shape;
      for (int i = 0; i < axis; i++) {
        tmp_out_shape.emplace_back(in_dims[i]);
      }
      tmp_out_shape.emplace_back(1);
      for (int i = axis + 1; i < in_dims.size(); i++) {
        tmp_out_shape.emplace_back(in_dims[i]);
      }
      DDim tmp_out_dims = phi::make_ddim(tmp_out_shape);
      output->Resize(tmp_out_dims);
      indices->Resize(tmp_out_dims);
    }
    DDim trans_dims(in_dims);
    DDim trans_out_dims(in_dims);
    for (int i = 0; i < trans.size(); i++) {
      trans_dims[i] = in_dims[trans[i]];
      trans_out_dims[i] = in_dims[trans[i]];
    }
    trans_out_dims[in_dims.size() - 1] = 1;
    DenseTensor trans_input;
    trans_input.mutable_data<T>(trans_dims, dev_ctx.GetPlace());
    int ndims = trans.size();
    funcs::TransCompute<phi::GPUContext, T>(
        ndims, dev_ctx, x, &trans_input, trans);
    DenseTensor trans_ind, trans_out;
    trans_ind.mutable_data<int64_t>(trans_out_dims, dev_ctx.GetPlace());
    trans_out.mutable_data<T>(trans_out_dims, dev_ctx.GetPlace());
    const int64_t input_height =
        phi::product(phi::slice_ddim(trans_dims, 0, trans_dims.size() - 1));
    const int64_t input_width = trans_dims[trans_dims.size() - 1];
    PADDLE_ENFORCE_EQ(
        SortKthvalue<T>(dev_ctx,
                        &trans_input,
                        input_width,
                        input_height,
                        k,
                        &trans_out,
                        &trans_ind),
        true,
        phi::errors::External("KthvalueOP: Error when use cub sorting"));
    funcs::TransCompute<phi::GPUContext, int64_t>(
        ndims, dev_ctx, trans_ind, indices, trans);
    funcs::TransCompute<phi::GPUContext, T>(
        ndims, dev_ctx, trans_out, output, trans);
    if (!keepdim) {
      output->Resize(out_dims);
      indices->Resize(out_dims);
    }
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(kthvalue,
                   GPU,
                   ALL_LAYOUT,
                   phi::KthvalueKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
