#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/top_k_grad_kernel.h"

#include "paddle/fluid/operators/top_k_function_cuda.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

namespace ops = paddle::operators;

template <typename T, typename Context>
void TopkGradKernel(const Context& dev_ctx,
                    const DenseTensor& x,
                    const DenseTensor& indices,
                    const DenseTensor& out_grad,
                    const Scalar& k_scalar,
                    int axis,
                    bool largest,
                    bool sorted,
                    DenseTensor* x_grad) {
  const auto& in_dims = x.dims();
  const auto& out_dims = indices.dims();

  int k = k_scalar.to<int>();

  // get the real the axis and the k
  if (axis < 0) {
    axis += in_dims.size();
  }
  const int& raw_height = in_dims[axis];

  // allocate the cuda memory for the x_grad
  T* x_grad_data = dev_ctx.template Alloc<T>(x_grad);
  const T* out_grad_data = out_grad.data<T>();
  const int64_t* indices_data = indices.data<int64_t>();

  int pre, n, post;
  ops::GetDims(in_dims, axis, &pre, &n, &post);

  // calcluate the block and grid num
  auto ComputeBlockSize = [](int col) {
    if (col > 512)
      return 1024;
    else if (col > 256 && col <= 512)
      return 512;
    else if (col > 128 && col <= 256)
      return 256;
    else if (col > 64 && col <= 128)
      return 128;
    else
      return 64;
  };
  int block_size = ComputeBlockSize(post * k);
  int max_threads = dev_ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(((max_threads - 1) / block_size + 1), 1);
  int grid_size = std::min(max_blocks, pre);

  // lanuch the cuda kernel to assign the grad
  ops::AssignGradWithAxis<T>
      <<<grid_size, block_size, 64 * 4, dev_ctx.stream()>>>(
          out_grad_data, indices_data, x_grad_data, pre, post, n, k);
}

}  // namespace phi

PD_REGISTER_KERNEL(top_k_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::TopkGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
