#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/cross_entropy_grad_kernel.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/operators/math/cross_entropy.h"
#include "paddle/fluid/operators/math/softmax.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/core/visit_type.h"
#include "paddle/phi/kernels/funcs/axis_utils.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/gpudnn/softmax_gpudnn.h"

namespace phi {

template <typename T>
__global__ void SoftLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const T* loss_grad,
                                                    const T* labels,
                                                    const int n,
                                                    const int d,
                                                    const int remain) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int idx_n = ids / d;
    int idx_remain = ids % remain;
    int idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (-labels[ids] / logit_grad[ids]);
  }
}

template <typename T, typename LabelT>
__global__ void HardLabelCrossEntropyGradientKernel(T* logit_grad,
                                                    const LabelT* labels,
                                                    const int n,
                                                    const int d,
                                                    const int remain,
                                                    const int ignore_index) {
  CUDA_KERNEL_LOOP(index, n * remain) {
    int idx_n = index / remain;
    int idx_remain = index % remain;
    int tmp = static_cast<int>(labels[index]);
    int idx = idx_n * d + tmp * remain + idx_remain;
    if (ignore_index != tmp) {
      logit_grad[idx] = -static_cast<T>(1.) / logit_grad[idx];
    }
  }
}

template <typename T, typename LabelT>
__global__ void ScaleCrossEntropyGradient(T* logit_grad,
                                          const T* loss_grad,
                                          const int num,
                                          const int d,
                                          const int remain,
                                          const LabelT* labels,
                                          const int ignore_index) {
  CUDA_KERNEL_LOOP(index, num) {
    int idx_n = index / d;
    int idx_remain = index % remain;
    int idx_lbl = idx_n * remain + idx_remain;
    int k = (index % d) / remain;
    auto lbl = static_cast<int64_t>(labels[idx_lbl]);
    if (lbl == ignore_index || lbl != k) {
      logit_grad[index] = static_cast<T>(0.);
    } else {
      logit_grad[index] *= loss_grad[idx_lbl];
    }
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels,
                                               const int64_t n,
                                               const int64_t d,
                                               const int64_t remain) {
  int64_t ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < n * d) {
    int64_t idx_n = ids / d;
    int64_t idx_remain = ids % remain;
    int64_t idx_loss = idx_n * remain + idx_remain;
    logit_grad[ids] = loss_grad[idx_loss] * (logit_grad[ids] - labels[ids]);
  }
}

/*
  Wrapper of softmax with cross entropy grad hard label.
*/
template <typename T, typename LabelT>
__global__ void SoftmaxWithCrossEntropyGradHardLabel(T* logits_grad,
                                                     const T* loss_grad,
                                                     const T* softmax,
                                                     const LabelT* labels,
                                                     const int64_t n,
                                                     const int64_t dim,
                                                     const int64_t d,
                                                     const int ignore_index) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t idx_n = idx / (d * dim);
  int64_t idx_dim = (idx / d) % dim;
  int64_t idx_d = idx % d;
  int64_t ids = idx_n * d + idx_d;

  if (idx < n * dim * d) {
    auto lbl = static_cast<int64_t>(labels[ids]);
    if (lbl == ignore_index) {
      logits_grad[idx] = static_cast<T>(0.0);
    } else if (lbl == idx_dim) {
      logits_grad[idx] = (softmax[idx] - static_cast<T>(1.0)) * loss_grad[ids];
    } else {
      logits_grad[idx] = softmax[idx] * loss_grad[ids];
    }
  }
}

template <typename T, typename LabelT>
void CrossEntropyWithSoftmaxGradGPUKernel(const GPUContext& dev_ctx,
                                          const DenseTensor& label,
                                          const DenseTensor& softmax,
                                          const DenseTensor& loss_grad,
                                          bool soft_label,
                                          bool use_softmax,
                                          bool numeric_stable_mode,
                                          int ignore_index,
                                          int axis,
                                          DenseTensor* logits_grad) {
  PADDLE_ENFORCE_EQ(
      dev_ctx.GetPlace().GetType(),
      phi::AllocationType::GPU,
      phi::errors::Unavailable("softmax_with_cross_entropy operator's "
                               "CUDA kernel only runs on GPU device."));
  const T* loss_grad_data = loss_grad.data<T>();
  DenseTensor* logit_grad = logits_grad;

  T* logit_grad_data = nullptr;
  bool copy_flag = (logit_grad != &softmax && (!use_softmax || soft_label));
  if (copy_flag) {
    phi::Copy(dev_ctx, softmax, dev_ctx.GetPlace(), false, logit_grad);
    logit_grad_data = logit_grad->data<T>();
  } else {
    logit_grad_data = dev_ctx.template Alloc<T>(logit_grad);
  }

  const int rank = logit_grad->dims().size();
  const int axis_v = phi::funcs::CanonicalAxis(axis, rank);
  int axis_dim = logit_grad->dims()[axis_v];

  const int64_t n = phi::funcs::SizeToAxis(axis_v, logit_grad->dims());
  const int64_t d = phi::funcs::SizeFromAxis(axis_v, logit_grad->dims());
  const int64_t remain = d / axis_dim;

#ifdef __HIPCC__
  int block = 256;
#else
  int block = 512;
#endif
  auto stream = dev_ctx.stream();

  // do not with softmax op, and input is softmax
  if (!use_softmax) {
    if (soft_label) {
      int grid = (n * d + block - 1) / block;
      const T* label_data = label.data<T>();
      SoftLabelCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          logit_grad_data, loss_grad_data, label_data, n, d, remain);
    } else {
      DenseTensor logits_grad_2d(*logit_grad);
      logits_grad_2d.Resize({n, d});
      int grid = (n * remain + block - 1) / block;
      const auto* label_data = label.data<LabelT>();
      HardLabelCrossEntropyGradientKernel<T, LabelT>
          <<<grid, block, 0, stream>>>(
              logit_grad_data, label_data, n, d, remain, ignore_index);
      int num = n * d;
      grid = (num + block - 1) / block;
      ScaleCrossEntropyGradient<T, LabelT>
          <<<grid, block, 0, stream>>>(logit_grad_data,
                                       loss_grad_data,
                                       num,
                                       d,
                                       remain,
                                       label_data,
                                       ignore_index);
    }

    return;
  }

  // with softmax, continue

  if (soft_label) {
    int64_t grid = (n * d + block - 1) / block;
    const T* label_data = label.data<T>();
    SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
        logit_grad_data, loss_grad_data, label_data, n, d, remain);
  } else {
    const T* softmax_data = softmax.data<T>();
    const auto* label_data = label.data<LabelT>();
    int grid = (n * d + block - 1) / block;
    SoftmaxWithCrossEntropyGradHardLabel<T>
        <<<grid, block, 0, stream>>>(logit_grad_data,
                                     loss_grad_data,
                                     softmax_data,
                                     label_data,
                                     n,
                                     d / remain,
                                     remain,
                                     ignore_index);
  }
}

template <typename T, typename Context>
void CrossEntropyWithSoftmaxGradKernel(const Context& dev_ctx,
                                       const DenseTensor& label,
                                       const DenseTensor& softmax,
                                       const DenseTensor& loss_grad,
                                       bool soft_label,
                                       bool use_softmax,
                                       bool numeric_stable_mode,
                                       int ignore_index,
                                       int axis,
                                       DenseTensor* logits_grad) {
  auto dtype = label.dtype();
  if (soft_label) {
    PADDLE_ENFORCE_EQ(
        dtype,
        paddle::experimental::CppTypeToDataType<T>::Type(),
        phi::errors::InvalidArgument("The Input(Label) should be with the "
                                     "same data type as kernel data type."));
    CrossEntropyWithSoftmaxGradGPUKernel<T, T>(dev_ctx,
                                               label,
                                               softmax,
                                               loss_grad,
                                               soft_label,
                                               use_softmax,
                                               numeric_stable_mode,
                                               ignore_index,
                                               axis,
                                               logits_grad);
  } else {
    PD_VISIT_INTEGRAL_TYPES(
        dtype, "CrossEntropyWithSoftmaxGradGPUKernel", ([&] {
          CrossEntropyWithSoftmaxGradGPUKernel<T, data_t>(dev_ctx,
                                                          label,
                                                          softmax,
                                                          loss_grad,
                                                          soft_label,
                                                          use_softmax,
                                                          numeric_stable_mode,
                                                          ignore_index,
                                                          axis,
                                                          logits_grad);
        }));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(cross_entropy_with_softmax_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::CrossEntropyWithSoftmaxGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
