#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include "paddle/phi/kernels/norm_grad_kernel.h"
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/fluid/operators/amp/fp16_type_traits.h"
#include "paddle/phi/common/bfloat16.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

#include "paddle/phi/kernels/funcs/common_shape.h"

namespace phi {

template <typename T, int BlockDim>
__global__ void NormalizeGradient(const T* x,
                                  const T* x_norm,
                                  const T* y_grad,
                                  const int pre,
                                  const int axis_n,
                                  const int post,
                                  T* x_grad) {
  using MT = typename paddle::operators::details::MPTypeTrait<T>::Type;
  typedef hipcub::BlockReduce<MT, BlockDim> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage_sum;
  int num = pre * post;
  for (int i = blockIdx.x; i < num; i += gridDim.x) {
    MT sum = 0.0;
    __shared__ MT row_sum;
    __shared__ MT row_sqrt_norm;
    __shared__ MT row_norm;

    auto base = (i / post) * post * axis_n + (i % post);

    for (int j = threadIdx.x; j < axis_n; j += blockDim.x) {
      int index = base + j * post;
      sum += static_cast<MT>(x[index]) * static_cast<MT>(y_grad[index]);
    }
    MT reduce_result = BlockReduce(temp_storage_sum).Sum(sum);

    if (threadIdx.x == 0) {
      row_sum = reduce_result;
      row_sqrt_norm = static_cast<MT>(x_norm[i]);
      row_norm = row_sqrt_norm * row_sqrt_norm;
    }
    __syncthreads();
    for (int j = threadIdx.x; j < axis_n; j += blockDim.x) {
      int index = base + j * post;
      const MT x_ij = static_cast<MT>(x[index]);
      const MT dy_ij = static_cast<MT>(y_grad[index]);
      x_grad[index] =
          static_cast<T>((dy_ij - x_ij * row_sum / row_norm) / row_sqrt_norm);
    }
  }
}

template <typename T, typename Context>
void NormGradKernel(const Context& ctx,
                    const DenseTensor& out_grad,
                    const DenseTensor& x,
                    const DenseTensor& norm,
                    int axis,
                    float epsilon,
                    bool is_test,
                    DenseTensor* x_grad) {
  auto* in_x = &x;
  auto* in_norm = &norm;
  auto* in_dy = &out_grad;
  auto* out_dx = x_grad;
  ctx.template Alloc<T>(out_dx);
  T* dx = out_dx->data<T>();
  const T* x_data = in_x->data<T>();
  const T* x_norm = in_norm->data<T>();
  const T* dy = in_dy->data<T>();

  auto xdim = in_x->dims();
  if (axis < 0) axis = xdim.size() + axis;
  int pre, n, post;
  funcs::GetPrePostNumel(xdim, axis, &pre, &n, &post);

#ifdef __HIPCC__
  const int block = 256;
#else
  const int block = 512;
#endif
  int max_threads = ctx.GetMaxPhysicalThreadCount();
  const int max_blocks = std::max(max_threads / block, 1);
  int grid = std::min(max_blocks, pre * post);
  NormalizeGradient<T, block><<<grid, block, 0, ctx.stream()>>>(
      x_data, x_norm, dy, pre, n, post, dx);
}

}  // namespace phi

PD_REGISTER_KERNEL(norm_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::NormGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
