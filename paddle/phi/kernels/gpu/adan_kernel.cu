#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/adan_kernel.h"

#include <math.h>  // for sqrt in CPU and CUDA

#include <vector>

#include "glog/logging.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/for_range.h"

namespace phi {
template <typename T, typename TG, typename MT>
__global__ void AdanKernelREG(MT beta1,
                              MT beta2,
                              MT beta3,
                              MT epsilon,
                              MT weight_decay,
                              MT beta1_pow_,
                              MT beta2_pow_,
                              MT beta3_pow_,
                              bool no_prox,
                              const MT* moment1,
                              MT* moment1_out,
                              const MT* moment2,
                              MT* moment2_out,
                              const MT* moment3,
                              MT* moment3_out,
                              const MT* lr_,
                              const TG* grad,
                              const TG* pre_grad,
                              TG* pre_grad_out,
                              const T* param,
                              T* param_out,
                              const MT* master_param,
                              MT* master_param_out,
                              int ndim,
                              bool vanilla
                              ) {
  MT lr = *lr_;
  MT beta1_pow = beta1_pow_;
  MT beta2_pow = beta2_pow_;
  MT beta3_pow = beta3_pow_;
  MT one = static_cast<MT>(1.0);
  if (!vanilla)
  {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (; id < ndim; id += gridDim.x * blockDim.x) {
      MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
      MT g = static_cast<MT>(grad[id]);
      MT pre_g = static_cast<MT>(pre_grad[id]);
      MT g_diff = g - pre_g;
      MT update = g + beta2 * g_diff;

      MT mom1 = static_cast<MT>(moment1[id]);
      MT mom2 = static_cast<MT>(moment2[id]);
      MT mom3 = static_cast<MT>(moment3[id]);

      mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
      mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g_diff;
      mom3 = beta3 * mom3 + (static_cast<MT>(1.0) - beta3) * update * update;

      MT denom = (sqrt(mom3) / sqrt(one - beta3_pow)) + epsilon;
      update =
          (mom1 / (one - beta1_pow) + beta2 * mom2 / (one - beta2_pow)) / (denom);

      if (no_prox) {
        p = p * (one - lr * weight_decay) - update * lr;
      } else {
        p = p - (update * lr);
        p = p / (one + lr * weight_decay);
      }
      moment1_out[id] = mom1;
      moment2_out[id] = mom2;
      moment3_out[id] = mom3;
      pre_grad_out[id] = grad[id];
      param_out[id] = static_cast<T>(p);
      if (master_param_out) {
        master_param_out[id] = p;
      }
    }
  }else
  {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (; id < ndim; id += gridDim.x * blockDim.x) {
      MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
      MT g = static_cast<MT>(grad[id]);
      MT pre_g = static_cast<MT>(pre_grad[id]);
      MT g_diff = g - pre_g;
      MT update = g + beta2 * g_diff;

      MT mom1 = static_cast<MT>(moment1[id]);
      MT mom3 = static_cast<MT>(moment3[id]);

      mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g + beta2 * (1 - beta2) * g_diff;
      mom3 = beta3 * mom3 + (static_cast<MT>(1.0) - beta3) * update * update;

      MT denom = (sqrt(mom3) / sqrt(one - beta3_pow)) + epsilon;
      update = mom1 / (one - beta1_pow) / (denom);

      if (no_prox) {
        p = p * (one - lr * weight_decay) - update * lr;
      } else {
        p = p - (update * lr);
        p = p / (one + lr * weight_decay);
      }
      moment1_out[id] = mom1;
      moment3_out[id] = mom3;
      pre_grad_out[id] = grad[id];
      param_out[id] = static_cast<T>(p);
      if (master_param_out) {
        master_param_out[id] = p;
      }
    }
  }
}


template <typename T, typename TG, typename MT>
__global__ void AdanKernelMEM(MT beta1,
                              MT beta2,
                              MT beta3,
                              MT epsilon,
                              MT weight_decay,
                              const MT* beta1_pow_,
                              const MT* beta2_pow_,
                              const MT* beta3_pow_,
                              bool no_prox,
                              const MT* moment1,
                              MT* moment1_out,
                              const MT* moment2,
                              MT* moment2_out,
                              const MT* moment3,
                              MT* moment3_out,
                              const MT* lr_,
                              const TG* grad,
                              const TG* pre_grad,
                              TG* pre_grad_out,
                              const T* param,
                              T* param_out,
                              const MT* master_param,
                              MT* master_param_out,
                              int ndim,
                              bool vanilla) {
  MT lr = *lr_;
  MT beta1_pow = *beta1_pow_;
  MT beta2_pow = *beta2_pow_;
  MT beta3_pow = *beta3_pow_;
  MT one = static_cast<MT>(1.0);
  if (!vanilla)
  {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (; id < ndim; id += gridDim.x * blockDim.x) {
      MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
      MT g = static_cast<MT>(grad[id]);
      MT pre_g = static_cast<MT>(pre_grad[id]);
      MT g_diff = g - pre_g;
      MT update = g + beta2 * g_diff;

      MT mom1 = static_cast<MT>(moment1[id]);
      MT mom2 = static_cast<MT>(moment2[id]);
      MT mom3 = static_cast<MT>(moment3[id]);

      mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
      mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g_diff;
      mom3 = beta3 * mom3 + (static_cast<MT>(1.0) - beta3) * update * update;

      MT denom = (sqrt(mom3) / sqrt(one - beta3_pow)) + epsilon;
      update =
          (mom1 / (one - beta1_pow) + beta2 * mom2 / (one - beta2_pow)) / (denom);

      if (no_prox) {
        p = p * (one - lr * weight_decay) - update * lr;
      } else {
        p = p - (update * lr);
        p = p / (one + lr * weight_decay);
      }
      moment1_out[id] = mom1;
      moment2_out[id] = mom2;
      moment3_out[id] = mom3;
      pre_grad_out[id] = grad[id];
      param_out[id] = static_cast<T>(p);
      if (master_param_out) {
        master_param_out[id] = p;
      }
    }
  }else
  {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (; id < ndim; id += gridDim.x * blockDim.x) {
      MT p = master_param ? master_param[id] : static_cast<MT>(param[id]);
      MT g = static_cast<MT>(grad[id]);
      MT pre_g = static_cast<MT>(pre_grad[id]);
      MT g_diff = g - pre_g;
      MT update = g + beta2 * g_diff;

      MT mom1 = static_cast<MT>(moment1[id]);
      MT mom3 = static_cast<MT>(moment3[id]);

      mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g + beta2 * (1 - beta2) * g_diff;
      mom3 = beta3 * mom3 + (static_cast<MT>(1.0) - beta3) * update * update;

      MT denom = (sqrt(mom3) / sqrt(one - beta3_pow)) + epsilon;
      update = mom1 / (one - beta1_pow) / (denom);

      if (no_prox) {
        p = p * (one - lr * weight_decay) - update * lr;
      } else {
        p = p - (update * lr);
        p = p / (one + lr * weight_decay);
      }
      moment1_out[id] = mom1;
      moment3_out[id] = mom3;
      pre_grad_out[id] = grad[id];
      param_out[id] = static_cast<T>(p);
      if (master_param_out) {
        master_param_out[id] = p;
      }
    }
  }
  
}


template <typename T>
__global__ void UpdateBetaPow(T beta1,
                              T beta2,
                              T beta3,
                              const T* beta1_pow_,
                              const T* beta2_pow_,
                              const T* beta3_pow_,
                              T* beta1_pow_out,
                              T* beta2_pow_out,
                              T* beta3_pow_out) {
  *beta1_pow_out = beta1 * beta1_pow_[0];
  *beta2_pow_out = beta2 * beta2_pow_[0];
  *beta3_pow_out = beta3 * beta3_pow_[0];
}

template <typename T, typename Context>
void AdanDenseKernel(const Context& dev_ctx,
                     const DenseTensor& param,
                     const DenseTensor& grad,
                     const DenseTensor& learning_rate,
                     const DenseTensor& pre_grad,
                     const DenseTensor& moment1,
                     const paddle::optional<DenseTensor>& moment2,
                     const DenseTensor& moment3,
                     const DenseTensor& beta1_pow,
                     const DenseTensor& beta2_pow,
                     const DenseTensor& beta3_pow,
                     const paddle::optional<DenseTensor>& master_param,
                     const Scalar& beta1,
                     const Scalar& beta2,
                     const Scalar& beta3,
                     const Scalar& epsilon,
                     const Scalar& weight_decay,
                     bool no_prox,
                     bool multi_precision,
                     bool use_global_beta_pow,
                     bool vanilla,
                     DenseTensor* param_out,
                     DenseTensor* pre_grad_out,
                     DenseTensor* moment1_out,
                     DenseTensor* moment2_out,
                     DenseTensor* moment3_out,
                     DenseTensor* beta1_pow_out,
                     DenseTensor* beta2_pow_out,
                     DenseTensor* beta3_pow_out,
                     DenseTensor* master_param_outs) {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;
  const auto grad_type = grad.dtype();

  VLOG(4) << "use_global_beta_pow:" << use_global_beta_pow;
  VLOG(4) << "multi_precision:" << multi_precision;
  VLOG(4) << "no_prox:" << no_prox;
  VLOG(4) << "is_vanilla:" << vanilla;

  MPDType beta1_ = beta1.to<MPDType>();
  MPDType beta2_ = beta2.to<MPDType>();
  MPDType beta3_ = beta3.to<MPDType>();
  MPDType epsilon_ = epsilon.to<MPDType>();
  MPDType weight_decay_ = weight_decay.to<MPDType>();
  VLOG(3) << "beta1_pow.numel() : " << beta1_pow.numel()
          << "beta2_pow.numel() : " << beta2_pow.numel()
          << "beta3_pow.numel() : " << beta3_pow.numel();
  VLOG(3) << "param.numel(): " << param.numel();

  PADDLE_ENFORCE_EQ(
      beta1_pow_out->numel(),
      1,
      errors::InvalidArgument("beta1 pow output size should be 1, but received "
                              "value is:%d.",
                              beta1_pow_out->numel()));
  VLOG(3) << beta1_pow_out->numel();

  PADDLE_ENFORCE_EQ(
      beta2_pow_out->numel(),
      1,
      errors::InvalidArgument("beta2 pow output size should be 1, but received "
                              "value is:%d.",
                              beta2_pow_out->numel()));
  PADDLE_ENFORCE_EQ(
      beta3_pow_out->numel(),
      1,
      errors::InvalidArgument("beta3 pow output size should be 1, but received "
                              "value is:%d.",
                              beta3_pow_out->numel()));
  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision ? dev_ctx.template Alloc<MPDType>(master_param_outs)
                      : nullptr;

  const MPDType* moment2_in_data =
      vanilla ?  nullptr : moment2->data<MPDType>();
  MPDType* moment2_out_data =
      vanilla ?  nullptr : dev_ctx.template Alloc<MPDType>(moment2_out);

  // update param and moment
  int threads = 512;
  int blocks = (param.numel() + threads - 1) / threads;
  if (beta1_pow.place() == CPUPlace() && beta2_pow.place() == CPUPlace() && beta3_pow.place() == CPUPlace()){
    VLOG(3) << "beta_pow place in cpu";
    // Compute with betapow in REG
    if (grad_type == phi::DataType::FLOAT32) {
      VLOG(3) << "CPU: grad type FLOAT32";
        AdanKernelREG<T, float, MPDType>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(
          beta1_,
          beta2_,
          beta3_,
          epsilon_,
          weight_decay_,
          *beta1_pow.data<MPDType>(),
          *beta2_pow.data<MPDType>(),
          *beta3_pow.data<MPDType>(),
          no_prox,
          moment1.data<MPDType>(),
          dev_ctx.template Alloc<MPDType>(moment1_out),
          moment2_in_data,
          moment2_out_data,
          moment3.data<MPDType>(),
          dev_ctx.template Alloc<MPDType>(moment3_out),
          learning_rate.data<MPDType>(),
          grad.data<float>(),
          pre_grad.data<float>(),
          dev_ctx.template Alloc<float>(pre_grad_out),
          param.data<T>(),
          dev_ctx.template Alloc<T>(param_out),
          master_in_data,
          master_out_data,
          param.numel(),
          vanilla);
      }else
      {
        VLOG(3) << "CPU: grad type Not FLOAT32";
        AdanKernelREG<T, T, MPDType><<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1_,
            beta2_,
            beta3_,
            epsilon_,
            weight_decay_,
            *beta1_pow.data<MPDType>(),
            *beta2_pow.data<MPDType>(),
            *beta3_pow.data<MPDType>(),
            no_prox,
            moment1.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment1_out),
            moment2_in_data,
            moment2_out_data,
            moment3.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment3_out),
            learning_rate.data<MPDType>(),
            grad.data<T>(),
            pre_grad.data<T>(),
            dev_ctx.template Alloc<T>(pre_grad_out),
            param.data<T>(),
            dev_ctx.template Alloc<T>(param_out),
            master_in_data,
            master_out_data,
            param.numel(),
            vanilla
        );
      }
      if (!use_global_beta_pow) {
      // Cpu update
      dev_ctx.template HostAlloc<MPDType>(beta1_pow_out)[0] =
          beta1_ * beta1_pow.data<MPDType>()[0];
      dev_ctx.template HostAlloc<MPDType>(beta2_pow_out)[0] =
          beta2_ * beta2_pow.data<MPDType>()[0];
      dev_ctx.template HostAlloc<MPDType>(beta3_pow_out)[0] =
          beta3_ * beta3_pow.data<MPDType>()[0];
      }
      

  }else{
    VLOG(3) << "beta_pow place in GPU";
     if (grad_type == phi::DataType::FLOAT32) {
      VLOG(3) << "GPU: grad type FLOAT32";
      AdanKernelMEM<T, float, MPDType>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1_,
            beta2_,
            beta3_,
            epsilon_,
            weight_decay_,
            beta1_pow.data<MPDType>(),
            beta2_pow.data<MPDType>(),
            beta3_pow.data<MPDType>(),
            no_prox,
            moment1.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment1_out),
            moment2_in_data,
            moment2_out_data,
            moment3.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment3_out),
            learning_rate.data<MPDType>(),
            grad.data<float>(),
            pre_grad.data<float>(),
            dev_ctx.template Alloc<float>(pre_grad_out),
            param.data<T>(),
            dev_ctx.template Alloc<T>(param_out),
            master_in_data,
            master_out_data,
            param.numel(),
            vanilla

          );
      }else
      {
        VLOG(3) << "GPU: grad type not FLOAT32";
        AdanKernelMEM<T, T, MPDType>
          <<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1_,
            beta2_,
            beta3_,
            epsilon_,
            weight_decay_,
            beta1_pow.data<MPDType>(),
            beta2_pow.data<MPDType>(),
            beta3_pow.data<MPDType>(),
            no_prox,
            moment1.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment1_out),
            moment2_in_data,
            moment2_out_data,
            moment3.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment3_out),
            learning_rate.data<MPDType>(),
            grad.data<T>(),
            pre_grad.data<T>(),
            dev_ctx.template Alloc<T>(pre_grad_out),
            param.data<T>(),
            dev_ctx.template Alloc<T>(param_out),
            master_in_data,
            master_out_data,
            param.numel(),
            vanilla

          );
      }
      if (!use_global_beta_pow) {
      // Update with gpu
      UpdateBetaPow<MPDType><<<1, 1, 0, dev_ctx.stream()>>>(
          beta1_,
          beta2_,
          beta3_,
          beta1_pow.data<MPDType>(),
          beta2_pow.data<MPDType>(),
          beta3_pow.data<MPDType>(),
          dev_ctx.template Alloc<MPDType>(beta1_pow_out),
          dev_ctx.template Alloc<MPDType>(beta2_pow_out),
          dev_ctx.template Alloc<MPDType>(beta3_pow_out));
      }
      
  }

}


}  // namespace phi

PD_REGISTER_KERNEL(adan,
                   GPU,
                   ALL_LAYOUT,
                   phi::AdanDenseKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  // Skip beta1_pow, beta2_pow, beta3_pow data transform
  kernel->InputAt(7).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(8).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(9).SetBackend(phi::Backend::ALL_BACKEND);

  if (kernel_key.dtype() == phi::DataType::FLOAT16 ||
      kernel_key.dtype() == phi::DataType::BFLOAT16) {
    kernel->OutputAt(1).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(2).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(3).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(4).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(5).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(6).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(7).SetDataType(phi::DataType::FLOAT32);
    kernel->OutputAt(8).SetDataType(phi::DataType::FLOAT32);
  }
  kernel->OutputAt(5).SetBackend(phi::Backend::UNDEFINED);
  kernel->OutputAt(6).SetBackend(phi::Backend::UNDEFINED);
  kernel->OutputAt(7).SetBackend(phi::Backend::UNDEFINED);
}
