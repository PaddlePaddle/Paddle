#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/operators/utils.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/range_kernel.h"

namespace phi {

template <typename T>
__global__ void Range(T start, T step, int64_t size, T* out) {
  CUDA_KERNEL_LOOP(index, size) { out[index] = start + step * index; }
}

template <typename T, typename Context>
void RangeKernel(const Context& dev_ctx,
                 const DenseTensor& start,
                 const DenseTensor& end,
                 const DenseTensor& step,
                 DenseTensor* out) {
  T start_value = GetValue<T>(start);
  T end_value = GetValue<T>(end);
  T step_value = GetValue<T>(step);

  int64_t size = 0;
  GetSize(start_value, end_value, step_value, &size);
  out->Resize(phi::make_ddim({size}));
  T* out_data = dev_ctx.template Alloc<T>(out);

  int block = std::min(size, static_cast<int64_t>(256));
  int grid = (size + block - 1) / block;
  Range<T><<<grid, block, 0, dev_ctx.stream()>>>(
      start_value, step_value, size, out_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    range, GPU, ALL_LAYOUT, phi::RangeKernel, float, double, int64_t, int) {}
