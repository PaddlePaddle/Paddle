#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/index_sample_kernel.h"

#include <algorithm>
#include <vector>
#include "paddle/fluid/framework/convert_utils.h"
#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

namespace {
template <typename Context>
void LimitGridDim(const Context& ctx, dim3* grid_dim) {
  auto max_grid_dim =
      reinterpret_cast<const phi::GPUContext&>(ctx).GetCUDAMaxGridDimSize();
  grid_dim->x = grid_dim->x < max_grid_dim[0] ? grid_dim->x : max_grid_dim[0];
  grid_dim->y = grid_dim->y < max_grid_dim[1] ? grid_dim->y : max_grid_dim[1];
}
#define PREDEFINED_BLOCK_SIZE_X 512
#define PREDEFINED_BLOCK_SIZE 1024
#define MIN(a, b) ((a) < (b) ? (a) : (b))
}

template <typename T, typename IndexT = int>
__global__ void IndexSampleForward(const IndexT* index,
                                   const T* in_data,
                                   T* out_data,
                                   size_t index_length,
                                   size_t input_length,
                                   size_t batch_size) {
  unsigned int index_i = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int index_j = blockDim.y * blockIdx.y + threadIdx.y;
  for (; index_j < batch_size; index_j += blockDim.y * gridDim.y) {
    index_i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; index_i < index_length; index_i += blockDim.x * gridDim.x) {
      unsigned int index_idx = index_j * index_length + index_i;
      unsigned int in_idx = index_j * input_length + index_i;
      IndexT sample_idx = index[index_idx];
      out_data[index_idx] = in_data[in_idx - index_i + sample_idx];
    }
  }
}

template <typename T, typename Context>
void IndexSampleKernel(const Context& ctx,
                       const DenseTensor& x,
                       const DenseTensor& index,
                       DenseTensor* out) {
  auto index_type = index.dtype();
  bool index_type_match =
      index_type == DataType::INT32 || index_type == DataType::INT64;
  PADDLE_ENFORCE_EQ(
      index_type_match,
      true,
      errors::InvalidArgument(
          "Input(Index) holds the wrong type, it holds %s, but "
          "desires to be %s or %s",
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType(index_type)),
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType(DataType::INT32)),
          paddle::framework::DataTypeToString(
              paddle::framework::TransToProtoVarType((DataType::INT64)))));
  const T* in_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(out);
  auto stream = reinterpret_cast<const phi::GPUContext&>(ctx).stream();
  auto input_dim = x.dims();
  auto index_dim = index.dims();
  size_t batch_size = input_dim[0];
  size_t input_length = input_dim[1];
  size_t index_length = index_dim[1];

  auto block_width = paddle::platform::RoundToPowerOfTwo(index_length);
  block_width = MIN(block_width, PREDEFINED_BLOCK_SIZE_X);
  int block_height =
      paddle::platform::RoundToPowerOfTwo(index_length * batch_size) /
      block_width;
  block_height = MIN(block_height, PREDEFINED_BLOCK_SIZE / block_width);
  dim3 block_dim(block_width, block_height);
  dim3 grid_dim((index_length + block_dim.x - 1) / block_dim.x,
                (batch_size + block_dim.y - 1) / block_dim.y);
  LimitGridDim(ctx, &grid_dim);

  if (index_type == DataType::INT64) {
    const int64_t* index_data = index.data<int64_t>();
    IndexSampleForward<T, int64_t><<<grid_dim, block_dim, 0, stream>>>(
        index_data, in_data, out_data, index_length, input_length, batch_size);
  } else if (index_type == DataType::INT32) {
    const int* index_data = index.data<int>();
    IndexSampleForward<T, int><<<grid_dim, block_dim, 0, stream>>>(
        index_data, in_data, out_data, index_length, input_length, batch_size);
  }
}
}  // namespace phi

PD_REGISTER_KERNEL(index_sample,
                   GPU,
                   ALL_LAYOUT,
                   phi::IndexSampleKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
