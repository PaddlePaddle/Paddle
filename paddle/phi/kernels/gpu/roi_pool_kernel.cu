#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/roi_pool_kernel.h"

#include "paddle/fluid/memory/memory.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/place.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

static constexpr int kNumCUDAThreads = 512;
static constexpr int kNumMaxinumNumBlocks = 4096;

static inline int NumBlocks(const int N) {
  return std::min((N + kNumCUDAThreads - 1) / kNumCUDAThreads,
                  kNumMaxinumNumBlocks);
}

template <typename T>
__global__ void GPURoiPoolForward(const int nthreads,
                                  const T* input_data,
                                  const T* input_rois,
                                  const float spatial_scale,
                                  const int channels,
                                  const int height,
                                  const int width,
                                  const int pooled_height,
                                  const int pooled_width,
                                  int* box_batch_id_data,
                                  T* output_data,
                                  int64_t* arg_max_data) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (size_t i = index; i < nthreads; i += offset) {
    int pw = i % pooled_width;
    int ph = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % channels;
    int n = i / pooled_width / pooled_height / channels;

    const T* offset_input_rois = input_rois + n * kROISize;
    int box_batch_ind = box_batch_id_data[n];
    int box_start_w = round(offset_input_rois[0] * spatial_scale);
    int box_start_h = round(offset_input_rois[1] * spatial_scale);
    int box_end_w = round(offset_input_rois[2] * spatial_scale);
    int box_end_h = round(offset_input_rois[3] * spatial_scale);

    int box_width = max(box_end_w - box_start_w + 1, 1);
    int box_height = max(box_end_h - box_start_h + 1, 1);

    int hstart = static_cast<int>(
        floor(static_cast<double>(ph) * static_cast<double>(box_height) /
              static_cast<double>(pooled_height)));
    int wstart = static_cast<int>(
        floor(static_cast<double>(pw) * static_cast<double>(box_width) /
              static_cast<double>(pooled_width)));
    int hend = static_cast<int>(
        ceil(static_cast<double>(ph + 1) * static_cast<double>(box_height) /
             static_cast<double>(pooled_height)));
    int wend = static_cast<int>(
        ceil(static_cast<double>(pw + 1) * static_cast<double>(box_width) /
             static_cast<double>(pooled_width)));
    hstart = min(max(hstart + box_start_h, 0), height);
    hend = min(max(hend + box_start_h, 0), height);
    wstart = min(max(wstart + box_start_w, 0), width);
    wend = min(max(wend + box_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    T maxval = is_empty ? 0 : -std::numeric_limits<T>::max();
    int maxidx = -1;
    const T* offset_input_data =
        input_data + (box_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int input_data_index = h * width + w;
        if (offset_input_data[input_data_index] > maxval) {
          maxval = offset_input_data[input_data_index];
          maxidx = input_data_index;
        }
      }
    }
    output_data[i] = maxval;
    if (arg_max_data) {
      arg_max_data[i] = maxidx;
    }
  }
}

template <typename T, typename Context>
void RoiPoolKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& boxes,
                   const paddle::optional<DenseTensor>& boxes_num,
                   int pooled_height,
                   int pooled_width,
                   float spatial_scale,
                   DenseTensor* out,
                   DenseTensor* arg_max) {
  auto x_dims = x.dims();
  int batch_size = x_dims[0];
  auto in_stride = phi::stride(x_dims);
  int channels = x_dims[1];
  int height = x_dims[2];
  int width = x_dims[3];

  int rois_num = boxes.dims()[0];

  if (rois_num == 0) return;

  int output_size = out->numel();
  int blocks = NumBlocks(output_size);
  int threads = kNumCUDAThreads;

  DenseTensor box_batch_id_list;
  box_batch_id_list.Resize({rois_num});
  int* box_batch_id_data = dev_ctx.template HostAlloc<int>(&box_batch_id_list);
  auto gplace = dev_ctx.GetPlace();

  if (boxes_num) {
    int boxes_batch_size = boxes_num->numel();
    PADDLE_ENFORCE_EQ(
        boxes_batch_size,
        batch_size,
        phi::errors::InvalidArgument(
            "The batch size of input(ROIs) and input(X) must be the same but "
            "received batch size of input(ROIs) and input(X) is %d and %d "
            "respectively.",
            boxes_batch_size,
            batch_size));
    std::vector<int> boxes_num_list(boxes_batch_size);
    paddle::memory::Copy(phi::CPUPlace(),
                         boxes_num_list.data(),
                         gplace,
                         boxes_num->data<int>(),
                         sizeof(int) * boxes_batch_size,
                         0);
    int start = 0;
    for (int n = 0; n < boxes_batch_size; ++n) {
      for (int i = start; i < start + boxes_num_list[n]; ++i) {
        box_batch_id_data[i] = n;
      }
      start += boxes_num_list[n];
    }
  } else {
    auto boxes_lod = boxes.lod().back();
    int boxes_batch_size = boxes_lod.size() - 1;
    PADDLE_ENFORCE_EQ(
        boxes_batch_size,
        batch_size,
        phi::errors::InvalidArgument(
            "The batch size of input(ROIs) and input(X) must be the same but "
            "received batch size of input(ROIs) and input(X) is %d and %d "
            "respectively.",
            boxes_batch_size,
            batch_size));

    int boxes_num_with_lod = boxes_lod[boxes_batch_size];
    PADDLE_ENFORCE_EQ(rois_num,
                      boxes_num_with_lod,
                      phi::errors::InvalidArgument(
                          "The number of rois from input(ROIs) and its LOD "
                          "must be the same. Received rois %d of input(ROIs) "
                          "but the number of rois %d from its LOD is %d",
                          rois_num,
                          boxes_num_with_lod));
    for (int n = 0; n < boxes_batch_size; ++n) {
      for (size_t i = boxes_lod[n]; i < boxes_lod[n + 1]; ++i) {
        box_batch_id_data[i] = n;
      }
    }
  }

  int bytes = box_batch_id_list.numel() * sizeof(int);
  auto box_ptr = paddle::memory::Alloc(
      dev_ctx.GetPlace(),
      bytes,
      phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));
  int* box_id_data = reinterpret_cast<int*>(box_ptr->ptr());
  paddle::memory::Copy(gplace,
                       box_id_data,
                       phi::CPUPlace(),
                       box_batch_id_data,
                       bytes,
                       dev_ctx.stream());

  T* output_data = dev_ctx.template Alloc<T>(out);
  int64_t* arg_max_data = dev_ctx.template Alloc<int64_t>(arg_max);

  GPURoiPoolForward<T>
      <<<blocks, threads, 0, dev_ctx.stream()>>>(output_size,
                                                 x.data<T>(),
                                                 boxes.data<T>(),
                                                 spatial_scale,
                                                 channels,
                                                 height,
                                                 width,
                                                 pooled_height,
                                                 pooled_width,
                                                 box_id_data,
                                                 output_data,
                                                 arg_max_data);
}

}  // namespace phi

PD_REGISTER_KERNEL(
    roi_pool, GPU, ALL_LAYOUT, phi::RoiPoolKernel, float, double) {
  kernel->OutputAt(1).SetDataType(phi::DataType::INT64);
}
