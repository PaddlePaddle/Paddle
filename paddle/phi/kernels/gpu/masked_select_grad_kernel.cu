#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/scan.h>

#include "paddle/phi/backends/cpu/cpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/masked_select_grad_kernel.h"

namespace phi {

__global__ void SetMaskArrayT(const bool* mask, int32_t* mask_array, int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx])
      mask_array[idx] = 1;
    else
      mask_array[idx] = 0;
  }
}

template <typename T>
__global__ void SelectGradWithPrefixMask(const int32_t* mask_prefix_sum,
                                         const bool* mask,
                                         const T* input,
                                         T* out,
                                         int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  for (; idx < size; idx += blockDim.x * gridDim.x) {
    if (mask[idx]) {
      int index = mask_prefix_sum[idx];
      out[idx] = input[index];
    } else {
      out[idx] = 0;
    }
  }
}

template <typename T, typename Context>
void MaskedSelectGradKernel(const Context& dev_ctx,
                            const DenseTensor& out_grad,
                            const DenseTensor& x,
                            const DenseTensor& mask,
                            DenseTensor* x_grad) {
  auto* mask_data = mask.data<bool>();
  auto* input_data = out_grad.data<T>();
  auto* out_data = x_grad->mutable_data<T>(dev_ctx.GetPlace());

  auto input_size = out_grad.numel();
  auto mask_size = mask.numel();
  auto mask_dim = mask.dims();

  auto out_size = mask_size;

  DenseTensor mask_array;
  DenseTensor mask_prefix_sum;
  mask_array.Resize(mask_dim);
  mask_prefix_sum.Resize(mask_dim);

  int32_t* mask_array_data =
      mask_array.mutable_data<int32_t>(dev_ctx.GetPlace());
  int32_t* mask_prefix_sum_data =
      mask_prefix_sum.mutable_data<int32_t>(dev_ctx.GetPlace());
  int threads = 512;
  int grid = (mask_size + threads - 1) / threads;
  auto stream = dev_ctx.stream();
  SetMaskArrayT<<<grid, threads, 0, stream>>>(
      mask_data, mask_array_data, mask_size);

  thrust::device_ptr<int32_t> mask_array_dev_ptr =
      thrust::device_pointer_cast(mask_array_data);
  thrust::device_vector<int32_t> mask_array_vec(mask_array_dev_ptr,
                                                mask_array_dev_ptr + mask_size);
  thrust::exclusive_scan(thrust::device,
                         mask_array_vec.begin(),
                         mask_array_vec.end(),
                         mask_prefix_sum_data);

  SelectGradWithPrefixMask<T><<<grid, threads, 0, stream>>>(
      mask_prefix_sum_data, mask_data, input_data, out_data, mask_size);
}

}  // namespace phi

PD_REGISTER_KERNEL(masked_select_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaskedSelectGradKernel,
                   float,
                   double,
                   int,
                   int64_t) {}
