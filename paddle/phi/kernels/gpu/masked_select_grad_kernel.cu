// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/masked_select_grad_kernel.h"

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/reverse.h>
#include <thrust/scan.h>

#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/expand_grad_kernel.h"
#include "paddle/phi/kernels/expand_kernel.h"
#include "paddle/phi/kernels/funcs/common_shape.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/funcs/select_impl.cu.h"

namespace phi {

template <typename MT, typename InT, typename OutT>
struct MaskedSelectGradFunctor {
  HOSTDEVICE MaskedSelectGradFunctor() = default;

  HOSTDEVICE inline void operator()(OutT* out,
                                    const MT* mask,
                                    const InT* value,
                                    int num) {
    int read_fix = 0;
    for (int idx = 0; idx < num; idx++) {
      if (mask[idx]) {
        out[idx] = value[read_fix++];
      } else {
        out[idx] = 0;
      }
    }
  }
};

template <typename T, typename Context>
void MaskedSelectGradKernel(const Context& dev_ctx,
                            const DenseTensor& x,
                            const DenseTensor& mask,
                            const DenseTensor& out_grad,
                            DenseTensor* x_grad) {
  // x_grad.size() == x.size()
  // x.size() == mask.size(), no broadcast, expand_mask = false, expand_x =
  // false x.size() < mask.size(), x broadcast to mask, expand_mask = false,
  // expand_x = true x.size() > mask.size(), mask broadcast to x, epxand_mask =
  // true, expand_x = false
  DenseTensor mask_expand;
  DenseTensor x_grad_expand;
  bool expand_x = false;

  auto expanded_size = funcs::MatrixGetBroadcastBatchPortion(
      common::vectorize(x_grad->dims()), common::vectorize(mask.dims()));
  auto expaned_dims = common::make_ddim(expanded_size);

  if (mask.dims() != expaned_dims) {
    ExpandKernel<bool, Context>(
        dev_ctx, mask, IntArray(expanded_size), &mask_expand);
  } else {
    mask_expand = mask;
  }

  if (x_grad->dims() != expaned_dims) {
    x_grad_expand = Empty<T, Context>(dev_ctx, IntArray(expanded_size));
    expand_x = true;
  } else {
    expand_x = false;
  }

  dev_ctx.template Alloc<T>(x_grad);
  auto mask_size = mask_expand.numel();
  if (mask_size <= 0) return;

  using Functor = MaskedSelectGradFunctor<bool, T, T>;

  DenseTensor* x_grad_tmp = x_grad;
  if (expand_x) {
    x_grad_tmp = &x_grad_expand;
  }

  phi::funcs::SelectKernel<bool, T, T, 2, Functor>(
      dev_ctx, mask_expand, out_grad, x_grad_tmp, Functor());

  if (expand_x) {
    ExpandGradKernel<T, Context>(
        dev_ctx, x, x_grad_expand, IntArray(expanded_size), x_grad);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(masked_select_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::MaskedSelectGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
