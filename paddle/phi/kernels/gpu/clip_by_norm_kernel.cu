#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/clip_by_norm_kernel.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/impl/clip_by_norm_kernel_impl.h"

#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"

namespace phi {

template <>
void ClipByNormKernel<phi::dtype::float16, phi::GPUContext>(
    const GPUContext& dev_ctx,
    const DenseTensor& x_in,
    float max_norm,
    DenseTensor* out_p) {
  dev_ctx.template Alloc<dtype::float16>(out_p);
  std::vector<int> reduce_dims;
  reduce_dims.resize(x_in.dims().size());
  for (int i = 0; i < reduce_dims.size(); ++i) {
    reduce_dims[i] = i;
  }

  DenseTensor tmp;
  tmp.Resize({1});
  dev_ctx.template Alloc<float>(&tmp);

  phi::funcs::ReduceKernel<dtype::float16,
                           float,
                           kps::AddFunctor,
                           kps::SquareFunctor<dtype::float16, float>>(
      dev_ctx,
      x_in,
      &tmp,
      kps::SquareFunctor<dtype::float16, float>(),
      reduce_dims);

  auto tmp_eigen = EigenVector<float>::Flatten(tmp);
  auto x_norm = tmp_eigen.sqrt();

  auto x = EigenVector<dtype::float16>::Flatten(x_in);
  auto out = EigenVector<dtype::float16>::Flatten(*out_p);

  auto& place = *dev_ctx.eigen_device();

  auto temp = (x_norm <= max_norm).template cast<float>();
  auto epsilon =
      ((x_norm <= static_cast<float>(1e-30)).all().template cast<float>()) *
      static_cast<float>(1e-6);

  auto scaling =
      (temp + (static_cast<float>(1) - temp) * max_norm / (x_norm + epsilon))
          .template cast<dtype::float16>();
  Eigen::array<int, 1> one_dim{{1}};
  Eigen::DSizes<int, 1> m_dsize(x_in.numel());

  out.device(place) = x * scaling.reshape(one_dim).broadcast(m_dsize);
}

template <>
void ClipByNormSparseKernel<phi::dtype::float16, phi::GPUContext>(
    const phi::GPUContext& ctx,
    const SelectedRows& x,
    float max_norm,
    SelectedRows* out) {
  // merge ids in selected rows first
  paddle::operators::math::scatter::MergeAdd<GPUContext, dtype::float16>
      merge_func;
  phi::SelectedRows merged_input;
  merge_func(ctx, x, &merged_input);
  auto input = merged_input.value();

  phi::SelectedRows* output_selected_rows = out;
  output_selected_rows->set_rows(merged_input.rows());
  output_selected_rows->set_height(merged_input.height());
  auto output = output_selected_rows->mutable_value();
  output->Resize(merged_input.value().dims());
  output->mutable_data<dtype::float16>(ctx.GetPlace());

  ClipByNormKernel<dtype::float16>(ctx, input, max_norm, output);
}

}  // namespace phi

// PD_REGISTER_KERNEL(
//     clip_by_norm, GPU, ALL_LAYOUT, phi::ClipByNormKernel, float,
//     phi::dtype::float16) {}

// PD_REGISTER_KERNEL(
//     clip_by_norm_sparse, GPU, ALL_LAYOUT, phi::ClipByNormSparseKernel, float,
//     phi::dtype::float16) {}
PD_REGISTER_KERNEL(
    clip_by_norm, GPU, ALL_LAYOUT, phi::ClipByNormKernel, phi::dtype::float16) {
}

PD_REGISTER_KERNEL(
    clip_by_norm_sparse, GPU, ALL_LAYOUT, phi::ClipByNormSparseKernel, float) {}
