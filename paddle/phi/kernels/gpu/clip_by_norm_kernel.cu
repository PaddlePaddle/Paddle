#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/clip_by_norm_kernel.h"

#include <typeinfo>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"
#include "paddle/phi/kernels/impl/clip_by_norm_kernel_impl.h"

namespace phi {

template <typename T, typename Context>
void ClipByNormKernel(const Context& dev_ctx,
                      const DenseTensor& in,
                      float max_norm,
                      DenseTensor* output) {
  if (typeid(T) == typeid(float)) {
    return ClipByNormFunctor<float, Context>(dev_ctx, in, max_norm, output);
  }
  auto input = &in;
  dev_ctx.template Alloc<dtype::float16>(output);

  PADDLE_ENFORCE_NOT_NULL(input,
                          phi::errors::InvalidArgument(
                              "Input(X) of ClipByNormOp should not be null. "
                              "Please check if it is created correctly."));
  std::vector<int> reduce_dims;
  reduce_dims.resize(input->dims().size());
  for (int i = 0; i < reduce_dims.size(); ++i) {
    reduce_dims[i] = i;
  }
  DenseTensor tmp_tensor;
  auto* tmp = &tmp_tensor;
  tmp->Resize({1});
  dev_ctx.template Alloc<float>(tmp);
  phi::funcs::ReduceKernel<dtype::float16,
                           float,
                           kps::AddFunctor,
                           kps::SquareFunctor<dtype::float16, float>>(
      dev_ctx,
      *input,
      tmp,
      kps::SquareFunctor<dtype::float16, float>(),
      reduce_dims);
  auto tmp_eigen = phi::EigenVector<float>::Flatten(*tmp);
  auto x_norm = tmp_eigen.sqrt();

  auto x = phi::EigenVector<dtype::float16>::Flatten(*input);
  auto out = phi::EigenVector<dtype::float16>::Flatten(*output);
  auto* place = dev_ctx.eigen_device();

  auto temp = (x_norm <= max_norm).template cast<float>();
  auto epsilon =
      ((x_norm <= static_cast<float>(1e-30)).all().template cast<float>()) *
      static_cast<float>(1e-6);

  auto scaling =
      (temp + (static_cast<float>(1) - temp) * max_norm / (x_norm + epsilon))
          .template cast<dtype::float16>();
  Eigen::array<int, 1> one_dim{{1}};
  Eigen::DSizes<int, 1> m_dsize(input->numel());

  out.device(*place) = x * scaling.reshape(one_dim).broadcast(m_dsize);
}

}  // namespace phi

PD_REGISTER_KERNEL(clip_by_norm,
                   GPU,
                   ALL_LAYOUT,
                   phi::ClipByNormKernel,
                   float,
                   phi::dtype::float16) {}
