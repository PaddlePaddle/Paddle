#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/truncated_gaussian_random_kernel.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <limits>

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"

#include "paddle/fluid/framework/generator.h"
// #include "paddle/phi/core/generator.h"

namespace phi {

template <typename T>
struct GPUTruncatedNormal {
  T mean, std;
  T a_normal_cdf;
  T b_normal_cdf;
  unsigned int seed;
  T numeric_min;

  __host__ __device__ GPUTruncatedNormal(T mean, T std, T numeric_min, int seed)
      : mean(mean), std(std), seed(seed), numeric_min(numeric_min) {
    a_normal_cdf = (1.0 + erff(-2.0 / sqrtf(2.0))) / 2.0;
    b_normal_cdf = (1.0 + erff(2.0 / sqrtf(2.0))) / 2.0;
  }

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed);
    thrust::uniform_real_distribution<T> dist(numeric_min, 1);
    rng.discard(n);
    T value = dist(rng);
    auto p = a_normal_cdf + (b_normal_cdf - a_normal_cdf) * value;
    return std::sqrt(2.0) * erfinvf(2 * p - 1) * std + mean;
  }
};

template <typename T>
struct TruncatedNormalOffset {
  T mean, std;
  T a_normal_cdf;
  T b_normal_cdf;
  unsigned int seed;
  T numeric_min;
  int offset_;

  __host__ __device__
  TruncatedNormalOffset(T mean, T std, T numeric_min, int seed, int offset)
      : mean(mean),
        std(std),
        seed(seed),
        numeric_min(numeric_min),
        offset_(offset) {
    a_normal_cdf = (1.0 + erff(-2.0 / sqrtf(2.0))) / 2.0;
    b_normal_cdf = (1.0 + erff(2.0 / sqrtf(2.0))) / 2.0;
  }

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed);
    thrust::uniform_real_distribution<T> dist(numeric_min, 1);
    rng.discard(n + offset_);
    T value = dist(rng);
    auto p = a_normal_cdf + (b_normal_cdf - a_normal_cdf) * value;
    return std::sqrt(2.0) * erfinvf(2 * p - 1) * std + mean;
  }
};

template <typename T, typename Context>
void TruncatedGaussianRandomKernel(const Context& dev_ctx,
                                   const ScalarArray& shape,
                                   float mean,
                                   float std,
                                   int seed,
                                   DataType dtype,
                                   DenseTensor* out) {
  auto tensor = out;

  T* data = dev_ctx.template Alloc<T>(tensor);

  bool seed_flag = false;
  if (seed == 0) {
    std::random_device rd;
    seed = rd();
    seed_flag = true;
  }

  thrust::counting_iterator<int64_t> index_sequence_begin(0);
  int64_t size = tensor->numel();

  int device_id = dev_ctx.GetPlace().GetDeviceId();
  auto gen_cuda = paddle::framework::GetDefaultCUDAGenerator(device_id);

  if (gen_cuda->GetIsInitPy() && seed_flag) {
    auto seed_offset = gen_cuda->IncrementOffset(1);
    int64_t gen_offset = size * seed_offset.second;
    thrust::transform(index_sequence_begin,
                      index_sequence_begin + size,
                      thrust::device_ptr<T>(data),
                      TruncatedNormalOffset<T>(mean,
                                               std,
                                               std::numeric_limits<T>::min(),
                                               seed_offset.first,
                                               gen_offset));
  } else {
    thrust::transform(
        index_sequence_begin,
        index_sequence_begin + size,
        thrust::device_ptr<T>(data),
        GPUTruncatedNormal<T>(mean, std, std::numeric_limits<T>::min(), seed));
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(truncated_gaussian_random,
                   GPU,
                   ALL_LAYOUT,
                   phi::TruncatedGaussianRandomKernel,
                   float) {}
