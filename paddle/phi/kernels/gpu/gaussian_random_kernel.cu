// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/gaussian_random_kernel.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/distribution_helper.h"
#include "paddle/phi/kernels/funcs/index_impl.cu.h"

#include "paddle/fluid/framework/generator.h"

DECLARE_bool(use_curand);

namespace phi {

template <typename T>
struct GaussianGenerator {
  T mean_, std_;
  unsigned int seed_;
  unsigned int offset_ = 0;

  __host__ __device__ GaussianGenerator(T mean, T std, int seed)
      : mean_(mean), std_(std), seed_(seed) {}

  __host__ __device__ GaussianGenerator(T mean, T std, int seed, int offset)
      : mean_(mean), std_(std), seed_(seed), offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    using MT = typename phi::dtype::MPTypeTrait<T>::Type;
    thrust::normal_distribution<MT> dist(static_cast<MT>(mean_),
                                         static_cast<MT>(std_));
    unsigned int new_n = n + offset_;
    rng.discard(new_n);
    MT out = dist(rng);
    return static_cast<T>(out);
  }
};

template <typename T, typename Context>
void GaussianRandomKernel(const Context& dev_ctx,
                          const IntArray& shape,
                          float mean,
                          float std,
                          int seed,
                          DataType dtype,
                          DenseTensor* out) {
  auto tensor = out;

  bool seed_flag = false;
  if (seed == 0) {
    std::random_device rd;
    seed = rd();
    seed_flag = true;
  }

  tensor->Resize(phi::make_ddim(shape.GetData()));

  T* data = dev_ctx.template Alloc<T>(tensor);

  int64_t size = tensor->numel();

  int device_id = dev_ctx.GetPlace().GetDeviceId();
  auto gen_cuda = paddle::framework::GetDefaultCUDAGenerator(device_id);

  if (gen_cuda->GetIsInitPy() && seed_flag) {
    if (FLAGS_use_curand) {
      using MT = typename phi::dtype::MPTypeTrait<T>::Type;
      funcs::normal_distribution<MT> dist;
      funcs::normal_transform<MT> trans(static_cast<MT>(mean),
                                        static_cast<MT>(std));
      funcs::distribution_and_transform<T>(dev_ctx, tensor, dist, trans);
    } else {
      auto seed_offset = gen_cuda->IncrementOffset(1);
      int64_t gen_offset = size * seed_offset.second;
      auto func = GaussianGenerator<T>(static_cast<T>(mean),
                                       static_cast<T>(std),
                                       seed_offset.first,
                                       gen_offset);
      IndexKernel<T, GaussianGenerator<T>>(dev_ctx, tensor, func);
    }
  } else {
    auto func =
        GaussianGenerator<T>(static_cast<T>(mean), static_cast<T>(std), seed);
    IndexKernel<T, GaussianGenerator<T>>(dev_ctx, tensor, func);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(gaussian_random,
                   GPU,
                   ALL_LAYOUT,
                   phi::GaussianRandomKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   float,
                   double) {}
