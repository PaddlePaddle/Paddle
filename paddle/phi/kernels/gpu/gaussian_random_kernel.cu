// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/gaussian_random_kernel.h"

#include <thrust/random.h>

#include "paddle/fluid/framework/generator.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/distribution_helper.h"
#include "paddle/phi/kernels/funcs/index_impl.cu.h"

namespace phi {

template <typename T>
struct GaussianGenerator {
  T mean_, std_;
  unsigned int seed_;
  unsigned int offset_ = 0;

  __host__ __device__ GaussianGenerator(T mean, T std, int seed)
      : mean_(mean), std_(std), seed_(seed) {}

  __host__ __device__ GaussianGenerator(T mean, T std, int seed, int offset)
      : mean_(mean), std_(std), seed_(seed), offset_(offset) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    using MT = typename phi::dtype::MPTypeTrait<T>::Type;
    thrust::normal_distribution<MT> dist(static_cast<MT>(mean_),
                                         static_cast<MT>(std_));
    unsigned int new_n = n + offset_;
    rng.discard(new_n);
    MT out = dist(rng);
    return static_cast<T>(out);
  }
};

template <typename T, typename Context>
void GaussianRandomKernel(const Context& dev_ctx,
                          const IntArray& shape,
                          float mean,
                          float std,
                          int seed,
                          DataType dtype,
                          DenseTensor* out) {
  out->Resize(phi::make_ddim(shape.GetData()));
  dev_ctx.template Alloc<T>(out);
  if (seed == 0) {
    // use global Generator seed
    using MT = typename phi::dtype::MPTypeTrait<T>::Type;
    funcs::normal_distribution<MT> dist;
    funcs::normal_transform<MT> trans(static_cast<MT>(mean),
                                      static_cast<MT>(std));
    funcs::distribution_and_transform<T>(dev_ctx, out, dist, trans);
  } else {
    // use OP seed
    auto func =
        GaussianGenerator<T>(static_cast<T>(mean), static_cast<T>(std), seed);
    IndexKernel<T, GaussianGenerator<T>>(dev_ctx, out, func);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(gaussian_random,
                   GPU,
                   ALL_LAYOUT,
                   phi::GaussianRandomKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   float,
                   double) {}
