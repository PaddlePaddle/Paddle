#include "hip/hip_runtime.h"
/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/contiguous_kernel.h"

#include <set>

#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/transpose_kernel.h"

namespace phi {
bool VerifyThreadConfigurationParameters(const dim3& block, const dim3& grid) {
  return block.x <= 1024 && block.y <= 1024 && block.z <= 64 &&
         block.x * block.y * block.z <= 1024 &&
         block.x * block.y * block.z >= 96 && grid.y < 65536 && grid.z < 65536;
}

template <typename T, size_t N>
__global__ void ContiguousCaseZeroFunc(
    const T* input_data,
    T* out_data,
    Array<int64_t, phi::DDim::kMaxRank + 1> input_stride) {
  int64_t input_offset = 0;
  int64_t output_offset = (blockIdx.z * gridDim.y * gridDim.x +
                           blockIdx.y * gridDim.x + blockIdx.x) *
                              blockDim.z * blockDim.y * blockDim.x +
                          threadIdx.z * blockDim.y * blockDim.x +
                          threadIdx.y * blockDim.x + threadIdx.x;
  int64_t coordinate[6] = {threadIdx.x,
                           threadIdx.y,
                           threadIdx.z,
                           blockIdx.x,
                           blockIdx.y,
                           blockIdx.z};

#pragma unroll
  for (int dim = N - 1; dim >= 0; --dim) {
    input_offset += coordinate[N - 1 - dim] * input_stride[dim];
  }

  out_data[output_offset] = input_data[input_offset];
}

template <typename T, size_t N>
__global__ void ContiguousCaseOneFunc(
    const T* input_data,
    T* out_data,
    Array<int64_t, phi::DDim::kMaxRank + 1> input_stride,
    Array<int64_t, 6> dims,
    const int64_t x_max) {
  int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x < x_max) {
    int64_t input_offset = 0;
    int64_t output_offset = (blockIdx.z * gridDim.y + blockIdx.y) * x_max + x;

    int64_t reg_dims[6] = {
        dims[0], dims[1], dims[2], dims[3], dims[4], dims[5]};
    int64_t coordinate[phi::DDim::kMaxRank + 1];

    switch (N) {
      case 1:
        coordinate[0] = x % reg_dims[0];
        break;
      case 2:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        break;
      case 3:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        break;
      case 4:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        coordinate[3] = blockIdx.y % reg_dims[2];
        break;
      case 5:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        coordinate[3] = blockIdx.y % reg_dims[2];
        coordinate[4] = blockIdx.y / reg_dims[2] % reg_dims[3];
        break;
      case 6:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        coordinate[3] = blockIdx.y % reg_dims[2];
        coordinate[4] = blockIdx.y / reg_dims[2] % reg_dims[3];
        coordinate[5] = blockIdx.y / (reg_dims[2] * reg_dims[3]);
        break;
      case 7:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        coordinate[3] = blockIdx.y % reg_dims[2];
        coordinate[4] = blockIdx.y / reg_dims[2] % reg_dims[3];
        coordinate[5] = blockIdx.y / (reg_dims[2] * reg_dims[3]);
        coordinate[6] = blockIdx.z % reg_dims[4];
        break;
      case 8:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        coordinate[3] = blockIdx.y % reg_dims[2];
        coordinate[4] = blockIdx.y / reg_dims[2] % reg_dims[3];
        coordinate[5] = blockIdx.y / (reg_dims[2] * reg_dims[3]);
        coordinate[6] = blockIdx.z % reg_dims[4];
        coordinate[7] = blockIdx.z / reg_dims[4] % reg_dims[5];
        break;
      case 9:
        coordinate[0] = x % reg_dims[0];
        coordinate[1] = x / reg_dims[0] % reg_dims[1];
        coordinate[2] = x / (reg_dims[0] * reg_dims[1]);
        coordinate[3] = blockIdx.y % reg_dims[2];
        coordinate[4] = blockIdx.y / reg_dims[2] % reg_dims[3];
        coordinate[5] = blockIdx.y / (reg_dims[2] * reg_dims[3]);
        coordinate[6] = blockIdx.z % reg_dims[4];
        coordinate[7] = blockIdx.z / reg_dims[4] % reg_dims[5];
        coordinate[8] = blockIdx.z / (reg_dims[4] * reg_dims[5]);
        break;
    }

#pragma unroll
    for (int dim = N - 1; dim >= 0; --dim) {
      input_offset += coordinate[N - 1 - dim] * input_stride[dim];
    }

    out_data[output_offset] = input_data[input_offset];
  }
}

template <typename T, size_t N>
__global__ void ContiguousDefaultFunc(
    const T* input_data,
    phi::Array<int64_t, phi::DDim::kMaxRank + 1> input_stride,
    phi::Array<int64_t, phi::DDim::kMaxRank + 1> dims,
    const int64_t numel,
    T* out_data) {
  int64_t gid = blockIdx.x * blockDim.x + threadIdx.x;
#pragma unroll
  for (int64_t i = gid; i < numel; i += blockDim.x * gridDim.x) {
    int64_t input_offset = 0;
    int64_t index_tmp = i;
#pragma unroll
    for (int dim = N - 1; dim >= 0; --dim) {
      input_offset += index_tmp % dims[dim] * input_stride[dim];
      index_tmp = index_tmp / dims[dim];
    }

    out_data[i] = input_data[input_offset];
  }
}

bool is_only_transposed(const DDim& shape,
                        const DDim& stride,
                        uint64_t offset,
                        DDim& src_shape,           // NOLINT
                        DDim& src_stride,          // NOLINT
                        std::vector<int>& axis) {  // NOLINT
  if (offset != 0) {
    return false;
  }
  std::set<int> visited_idx;
  axis.resize(stride.size());
  for (int i = 0; i < stride.size(); i++) {
    int64_t max_num = 0;
    int max_idx = -1;
    for (int j = 0; j < stride.size(); j++) {
      if (visited_idx.count(j)) {
        continue;
      }
      if (stride[j] < 1) {
        return false;
      }
      if (stride[j] > max_num) {
        max_num = stride[j];
        max_idx = j;
      }
    }
    if (max_idx == -1) {
      return false;
    }
    if (i != 0 && src_stride[i - 1] == max_num) {
      return false;
    }
    visited_idx.insert(max_idx);
    src_stride[i] = max_num;
    src_shape[i] = shape[max_idx];
    axis[max_idx] = i;
  }

  if (DenseTensorMeta::calc_strides(src_shape) == src_stride) {
    return true;
  } else {
    return false;
  }
}

template <typename T, typename Context>
bool LaunchContiguousCazeZeroKernel(
    const Context& dev_ctx,
    const T* input_data,
    const phi::Array<int64_t, phi::DDim::kMaxRank + 1>& input_stride,
    const phi::Array<int64_t, phi::DDim::kMaxRank + 1>& input_dims,
    int rank,
    T* output_data) {
  if (rank > 6) {
    return false;
  }

  dim3 grid(1, 1, 1), block(1, 1, 1);

  if (rank >= 1) {
    block.x = input_dims[rank - 1];
  }

  if (rank >= 2) {
    block.y = input_dims[rank - 2];
  }

  if (rank >= 3) {
    block.z = input_dims[rank - 3];
  }

  if (rank >= 4) {
    grid.x = input_dims[rank - 4];
  }

  if (rank >= 5) {
    grid.y = input_dims[rank - 5];
  }

  if (rank >= 6) {
    grid.z = input_dims[rank - 6];
  }

  if (!VerifyThreadConfigurationParameters(block, grid)) {
    return false;
  }

  switch (rank) {
    case 1:
      ContiguousCaseZeroFunc<T, 1><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, output_data, input_stride);
      break;
    case 2:
      ContiguousCaseZeroFunc<T, 2><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, output_data, input_stride);
      break;
    case 3:
      ContiguousCaseZeroFunc<T, 3><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, output_data, input_stride);
      break;
    case 4:
      ContiguousCaseZeroFunc<T, 4><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, output_data, input_stride);
      break;
    case 5:
      ContiguousCaseZeroFunc<T, 5><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, output_data, input_stride);
      break;
    case 6:
      ContiguousCaseZeroFunc<T, 6><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, output_data, input_stride);
      break;
  }

  return true;
}

template <typename T, typename Context>
bool LaunchContiguousCazeOneKernel(
    const Context& dev_ctx,
    const T* input_data,
    const phi::Array<int64_t, phi::DDim::kMaxRank + 1>& input_stride,
    const phi::Array<int64_t, phi::DDim::kMaxRank + 1>& input_dims,
    int rank,
    int numel,
    T* output_data) {
  dim3 grid(1, 1, 1), block(1, 1, 1);
  phi::Array<int64_t, 6> cur_input_dims;
  block.x = 512;

  if (rank >= 1) {
    grid.x = (numel + block.x - 1) / block.x;
    cur_input_dims[0] = input_dims[rank - 1];
  }

  if (rank >= 2) {
    cur_input_dims[1] = input_dims[rank - 2];
  }

  if (rank >= 4) {
    grid.x =
        (input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3] +
         block.x - 1) /
        block.x;
    grid.y = input_dims[rank - 4];
    cur_input_dims[2] = input_dims[rank - 4];
  }

  if (rank >= 5) {
    grid.y = input_dims[rank - 4] * input_dims[rank - 5];
    cur_input_dims[2] = input_dims[rank - 4];
    cur_input_dims[3] = input_dims[rank - 5];
  }

  if (rank >= 6) {
    grid.y = input_dims[rank - 4] * input_dims[rank - 5] * input_dims[rank - 6];
  }

  if (rank >= 7) {
    grid.z = input_dims[rank - 7];
    cur_input_dims[4] = input_dims[rank - 7];
  }

  if (rank >= 8) {
    grid.z = input_dims[rank - 7] * input_dims[rank - 8];
    cur_input_dims[5] = input_dims[rank - 8];
  }

  if (rank >= 9) {
    grid.z = input_dims[rank - 7] * input_dims[rank - 8] * input_dims[rank - 9];
  }

  if (!VerifyThreadConfigurationParameters(block, grid)) {
    return false;
  }

  switch (rank) {
    case 1:
      ContiguousCaseOneFunc<T, 1>
          <<<grid, block, 0, dev_ctx.stream()>>>(input_data,
                                                 output_data,
                                                 input_stride,
                                                 cur_input_dims,
                                                 input_dims[rank - 1]);
      break;
    case 2:
      ContiguousCaseOneFunc<T, 2><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2]);
      break;
    case 3:
      ContiguousCaseOneFunc<T, 3><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    case 4:
      ContiguousCaseOneFunc<T, 4><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    case 5:
      ContiguousCaseOneFunc<T, 5><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    case 6:
      ContiguousCaseOneFunc<T, 6><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    case 7:
      ContiguousCaseOneFunc<T, 7><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    case 8:
      ContiguousCaseOneFunc<T, 8><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    case 9:
      ContiguousCaseOneFunc<T, 9><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data,
          output_data,
          input_stride,
          cur_input_dims,
          input_dims[rank - 1] * input_dims[rank - 2] * input_dims[rank - 3]);
      break;
    default:
      PADDLE_THROW(phi::errors::InvalidArgument(
          "The rank of input should be less than 9, but received %d.", rank));
  }

  return true;
}

template <typename T, typename Context>
void LaunchContiguousDefaultKernel(
    const Context& dev_ctx,
    const T* input_data,
    const phi::Array<int64_t, phi::DDim::kMaxRank + 1>& input_stride,
    const phi::Array<int64_t, phi::DDim::kMaxRank + 1>& input_dims,
    int rank,
    int numel,
    T* output_data) {
  int64_t block = 512;
  int64_t grid = (numel + block - 1) / block;

  switch (rank) {
    case 1:
      ContiguousDefaultFunc<T, 1><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 2:
      ContiguousDefaultFunc<T, 2><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 3:
      ContiguousDefaultFunc<T, 3><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 4:
      ContiguousDefaultFunc<T, 4><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 5:
      ContiguousDefaultFunc<T, 5><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 6:
      ContiguousDefaultFunc<T, 6><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 7:
      ContiguousDefaultFunc<T, 7><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 8:
      ContiguousDefaultFunc<T, 8><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    case 9:
      ContiguousDefaultFunc<T, 9><<<grid, block, 0, dev_ctx.stream()>>>(
          input_data, input_stride, input_dims, numel, output_data);
      break;
    default:
      PADDLE_THROW(phi::errors::InvalidArgument(
          "The rank of input should be less than 9, but received %d.", rank));
  }
}

template <typename T, typename Context>
void ContiguousKernel(const Context& dev_ctx,
                      const DenseTensor& input,
                      DenseTensor* out) {
  phi::DenseTensorMeta meta = input.meta();
  std::vector<int> axis;
  DDim src_stride = meta.strides;
  DDim src_shape = meta.dims;
  if (is_only_transposed(
          meta.dims, meta.strides, meta.offset, src_shape, src_stride, axis)) {
    meta.strides = meta.calc_strides(meta.dims);
    out->set_meta(meta);
    DenseTensor tmp_tensor = input;
    phi::DenseTensorMeta tmp_meta = meta;
    tmp_meta.strides = src_stride;
    tmp_meta.dims = src_shape;
    tmp_tensor.set_meta(tmp_meta);
    TransposeKernel<T, Context>(dev_ctx, tmp_tensor, axis, out);
    return;
  }

  meta.strides = meta.calc_strides(meta.dims);
  meta.offset = 0;
  out->set_meta(meta);

  const T* input_data = input.data<T>();
  T* output_data = dev_ctx.template Alloc<T>(out);
  int rank = input.dims().size();
  auto numel = input.numel();

  if (numel <= 0) {
    return;
  }

  Array<int64_t, phi::DDim::kMaxRank + 1> input_stride;
  Array<int64_t, phi::DDim::kMaxRank + 1> input_dims;
  for (int i = 0; i < input.dims().size(); i++) {
    input_dims[i] = input.dims()[i];
    input_stride[i] = input.strides()[i];
  }

  if (rank == 0) {
    rank = 1;
    input_dims[0] = numel;
    input_stride[0] = 1;
  }

  if (LaunchContiguousCazeZeroKernel<T, Context>(
          dev_ctx, input_data, input_stride, input_dims, rank, output_data)) {
  } else if (LaunchContiguousCazeOneKernel<T, Context>(dev_ctx,
                                                       input_data,
                                                       input_stride,
                                                       input_dims,
                                                       rank,
                                                       numel,
                                                       output_data)) {
  } else {
    LaunchContiguousDefaultKernel<T, Context>(dev_ctx,
                                              input_data,
                                              input_stride,
                                              input_dims,
                                              rank,
                                              numel,
                                              output_data);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(contiguous,
                   GPU,
                   ALL_LAYOUT,
                   phi::ContiguousKernel,
                   bool,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int32_t,
                   int64_t,
                   float,
                   double,
                   ::phi::dtype::float16,
                   ::phi::dtype::bfloat16,
                   ::phi::dtype::complex<float>,
                   ::phi::dtype::complex<double>) {}
