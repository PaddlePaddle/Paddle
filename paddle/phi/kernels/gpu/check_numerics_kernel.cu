#include "hip/hip_runtime.h"
/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/check_numerics_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/memory_utils.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/check_numerics_utils.h"
#include "paddle/phi/kernels/funcs/math_cuda_utils.h"

DECLARE_int32(check_nan_inf_level);

namespace phi {

static std::once_flag init_multi_gpu_op_var_map_flag;

// lazy init
static std::vector<
    std::unordered_map<std::string, phi::Allocator::AllocationPtr>>&
multi_op_var2gpu_str() {
  static std::vector<
      std::unordered_map<std::string, phi::Allocator::AllocationPtr>>
      _multi_op_var2gpu_str;
  return _multi_op_var2gpu_str;
}

static std::vector<std::mutex>& multi_op_var2gpu_str_mutex() {
  static std::vector<std::mutex> _multi_op_var2gpu_str_mutex;
  return _multi_op_var2gpu_str_mutex;
}

static void InitMultiGPUOpVarMap() {
  int dev_count = phi::backends::gpu::GetGPUDeviceCount();
  PADDLE_ENFORCE_GT(dev_count,
                    0,
                    phi::errors::NotFound(
                        "cuda device must > 0, now dev_count=%d", dev_count));

  // https://stackoverflow.com/questions/16465633/how-can-i-use-something-like-stdvectorstdmutex
  std::vector<std::unordered_map<std::string, phi::Allocator::AllocationPtr>>
      tmp_multi(dev_count);
  std::vector<std::mutex> tmp_multi_mutex(dev_count);

  multi_op_var2gpu_str().swap(tmp_multi);
  multi_op_var2gpu_str_mutex().swap(tmp_multi_mutex);
}

template <typename T>
__device__ __forceinline__ void PrintNanInfKernel(const T* value,
                                                  const size_t numel,
                                                  int print_num,
                                                  char* debug_info) {
  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ unsigned int nan_count, inf_count, num_count;
  if (threadIdx.x == 0) nan_count = inf_count = num_count = 0;
  __syncthreads;

  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    unsigned int count = 0;
    if (isnan(value[i])) {
      count = atomicAdd(&nan_count, 1);
    } else if (isinf(value[i])) {
      count = atomicAdd(&inf_count, 1);
    } else {
      count = atomicAdd(&num_count, 1);
    }
    // for cuda, print in every block
    if (count < print_num) {
      printf("numel:%lu idx:%lu value:%f\n",
             static_cast<uint64_t>(numel),
             static_cast<uint64_t>(i),
             static_cast<float>(value[i]));
    }
  }
  __syncthreads;

#ifdef __HIPCC__
  if (true && hipThreadIdx_x == 0) {
    printf("In block %d, there has %u,%u,%u nan,inf,num\n",
           hipBlockIdx_x,
           nan_count,
           inf_count,
           num_count);
#else
  if (true && threadIdx.x == 0) {
    printf("In block %d, there has %u,%u,%u nan,inf,num\n",
           blockIdx.x,
           nan_count,
           inf_count,
           num_count);
#endif
    PADDLE_ENFORCE(false, "===ERROR: in %s find nan or inf===", debug_info);
  }
}

// Resnet 2gpus speed test, no check 270 images/s, this check 229 images/s
template <typename T>
__global__ void CheckNanInfKernel(const T* value,
                                  const size_t numel,
                                  int print_num,
                                  char* debug_info) {
  /// step 1, judge wheater has nan or inf
  __shared__ volatile int has_nan_inf;
  if (threadIdx.x == 0) has_nan_inf = false;
  __syncthreads();

  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  T sum = static_cast<T>(0.0);
  // Todo(wangxi). simd speed up
  for (size_t i = tid; i < numel; i += blockDim.x * gridDim.x) {
    sum += (value[i] - value[i]);
  }

  if (isnan(sum) || isinf(sum)) has_nan_inf = true;
  __syncthreads();

  /// Note. different blocks may behave differently
  if (!has_nan_inf) return;

  PrintNanInfKernel(value, numel, print_num, debug_info);
}

template <typename T, int ReduceType>
__device__ T BlockReduce(T value) {
  __shared__ T shared_mem[1024];

  shared_mem[threadIdx.x] = value;
  __syncthreads();

  for (int stride = blockDim.x >> 1; stride > 0; stride = stride >> 1) {
    if (threadIdx.x < stride) {
      T value0 = shared_mem[threadIdx.x];
      T value1 = shared_mem[threadIdx.x + stride];
      T reduce_value;
      if (ReduceType == 0) {
        // max
        reduce_value = value0 > value1 ? value0 : value1;
      } else if (ReduceType == 1) {
        // min
        reduce_value = value0 < value1 ? value0 : value1;
      } else if (ReduceType == 2) {
        // sum
        reduce_value = value0 + value1;
      }
      shared_mem[threadIdx.x] = reduce_value;
    }

    if (stride > 16) {
      __syncthreads();
    }
  }

  __syncthreads();
  return shared_mem[0];
}

__device__ void BlockReduceNumNanInfAndWrite(const int64_t num_nan,
                                             const int64_t num_inf,
                                             const int64_t num_zero,
                                             int64_t offset,
                                             int64_t* num_nan_ptr,
                                             int64_t* num_inf_ptr,
                                             int64_t* num_zero_ptr) {
  int64_t block_num_nan = BlockReduce<int64_t, 2>(num_nan);
  int64_t block_num_inf = BlockReduce<int64_t, 2>(num_inf);
  int64_t block_num_zero = BlockReduce<int64_t, 2>(num_zero);

  if (threadIdx.x == 0) {
    num_nan_ptr[offset] = block_num_nan;
    num_inf_ptr[offset] = block_num_inf;
    num_zero_ptr[offset] = block_num_zero;
  }
}

template <
    typename T,
    std::enable_if_t<std::is_same<T, phi::dtype::complex<float>>::value ||
                         std::is_same<T, phi::dtype::complex<double>>::value,
                     bool> = true>
__device__ void BlockReduceMaxMinAndWrite(const T max_value,
                                          const T min_value,
                                          const T mean_value,
                                          int64_t offset,
                                          T* max_ptr,
                                          T* min_ptr,
                                          T* mean_ptr) {
  // TODO(Xreki): support complex
}

template <
    typename T,
    std::enable_if_t<!std::is_same<T, phi::dtype::complex<float>>::value &&
                         !std::is_same<T, phi::dtype::complex<double>>::value,
                     bool> = true>
__device__ void BlockReduceMaxMinAndWrite(const T max_value,
                                          const T min_value,
                                          const T mean_value,
                                          int64_t offset,
                                          T* max_ptr,
                                          T* min_ptr,
                                          T* mean_ptr) {
  if (max_ptr && min_ptr && mean_ptr) {
    __syncthreads();

    T block_max_value = phi::funcs::BlockReduceMax<T>(max_value, FINAL_MASK);
    T block_min_value = phi::funcs::BlockReduceMin<T>(min_value, FINAL_MASK);
    T block_mean_value = phi::funcs::BlockReduceSum<T>(mean_value, FINAL_MASK);

    if (threadIdx.x == 0) {
      max_ptr[offset] = block_max_value;
      min_ptr[offset] = block_min_value;
      mean_ptr[offset] = block_mean_value;
    }
  }
}

template <typename T, typename MT>
__global__ void FindNanInfAndBlockMaxMin(const T* value_ptr,
                                         const int64_t numel,
                                         int64_t* block_num_nan_ptr,
                                         int64_t* block_num_inf_ptr,
                                         int64_t* block_num_zero_ptr,
                                         MT* tensor_block_max_ptr,
                                         MT* tensor_block_min_ptr,
                                         MT* tensor_block_mean_ptr) {
  int64_t i = threadIdx.x + blockIdx.x * blockDim.x;

  int64_t num_nan = 0;
  int64_t num_inf = 0;
  int64_t num_zero = 0;

  MT max_value = static_cast<MT>(i < numel ? value_ptr[i] : value_ptr[0]);
  MT min_value = static_cast<MT>(i < numel ? value_ptr[i] : value_ptr[0]);
  MT mean_value = static_cast<MT>(0);
  for (; i < numel; i += blockDim.x * gridDim.x) {
    MT value = static_cast<MT>(value_ptr[i]);

    max_value = value > max_value ? value : max_value;
    min_value = value < min_value ? value : min_value;
    mean_value += value / static_cast<MT>(numel);

    if (isnan(value)) {
      num_nan += 1;
    } else if (isinf(value)) {
      num_inf += 1;
    }
    if (value == static_cast<MT>(0)) {
      num_zero += 1;
    }
  }

  BlockReduceNumNanInfAndWrite(num_nan,
                               num_inf,
                               num_zero,
                               blockIdx.x,
                               block_num_nan_ptr,
                               block_num_inf_ptr,
                               block_num_zero_ptr);

  BlockReduceMaxMinAndWrite<MT>(max_value,
                                min_value,
                                mean_value,
                                blockIdx.x,
                                tensor_block_max_ptr,
                                tensor_block_min_ptr,
                                tensor_block_mean_ptr);
}

template <typename T, typename MT>
__global__ void FindGlobalMaxMinAndPrint(const int64_t* block_num_nan_ptr,
                                         const int64_t* block_num_inf_ptr,
                                         const int64_t* block_num_zero_ptr,
                                         const MT* tensor_block_max_ptr,
                                         const MT* tensor_block_min_ptr,
                                         const MT* tensor_block_mean_ptr,
                                         const char* debug_info,
                                         int64_t numel,
                                         int64_t numel_max_min,
                                         int check_nan_inf_level) {
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    int64_t num_nan = 0;
    int64_t num_inf = 0;
    int64_t num_zero = 0;

    // numel_max_min <= 128
    for (int64_t i = 0; i < numel_max_min; ++i) {
      num_nan += block_num_nan_ptr[i];
      num_inf += block_num_inf_ptr[i];
      num_zero += block_num_zero_ptr[i];
    }

    MT max_value = static_cast<MT>(0);
    MT min_value = static_cast<MT>(0);
    MT mean_value = static_cast<MT>(0);
    if (tensor_block_max_ptr && tensor_block_min_ptr && tensor_block_mean_ptr) {
      max_value = tensor_block_max_ptr[0];
      min_value = tensor_block_min_ptr[0];
      mean_value = tensor_block_mean_ptr[0];

      // numel_max_min <= 128
      for (int64_t i = 1; i < numel_max_min; ++i) {
        MT tmp_max_value = tensor_block_max_ptr[i];
        MT tmp_min_value = tensor_block_min_ptr[i];
        MT tmp_mean_value = tensor_block_mean_ptr[i];

        max_value = tmp_max_value > max_value ? tmp_max_value : max_value;
        min_value = tmp_min_value < min_value ? tmp_min_value : min_value;
        mean_value += tmp_mean_value;
      }
    }

    phi::funcs::PrintForDifferentLevel<T, MT>(debug_info,
                                              numel,
                                              num_nan,
                                              num_inf,
                                              num_zero,
                                              max_value,
                                              min_value,
                                              mean_value,
                                              check_nan_inf_level);
  }
}

template <typename T>
inline std::string GetHintString(const std::string& op_type,
                                 const std::string& var_name,
                                 const phi::Place& place,
                                 int dev_id = -1) {
  std::string op_var =
      phi::funcs::GetCpuHintString<T>(op_type, var_name, place, dev_id);
  PADDLE_ENFORCE_EQ(
      (dev_id >= 0 && dev_id < multi_op_var2gpu_str_mutex().size()),
      true,
      phi::errors::OutOfRange("GPU dev_id must >=0 and < dev_count=%d",
                              multi_op_var2gpu_str_mutex().size()));
  return op_var;
}

template <typename T>
static char* GetGpuHintStringPtr(const phi::GPUContext& ctx,
                                 const std::string& op_type,
                                 const std::string& var_name,
                                 int dev_id) {
  std::string op_var =
      GetHintString<T>(op_type, var_name, ctx.GetPlace(), dev_id);
  char* gpu_str_ptr = nullptr;

  {
    auto& op_var2gpu_str_mutex = multi_op_var2gpu_str_mutex().at(dev_id);
    auto& op_var2gpu_str = multi_op_var2gpu_str().at(dev_id);

    std::lock_guard<std::mutex> guard(op_var2gpu_str_mutex);
    if (op_var2gpu_str.find(op_var) == op_var2gpu_str.end()) {  // insert
      auto gpu_str_tensor = phi::memory_utils::Alloc(
          ctx.GetPlace(),
          op_var.length() + 1,
          phi::Stream(reinterpret_cast<phi::StreamId>(ctx.stream())));
      gpu_str_ptr = reinterpret_cast<char*>(gpu_str_tensor->ptr());

      op_var2gpu_str.emplace(op_var, std::move(gpu_str_tensor));

      auto iter = op_var2gpu_str.find(op_var);
      PADDLE_ENFORCE_EQ(iter != op_var2gpu_str.end(),
                        true,
                        phi::errors::PreconditionNotMet(
                            "op_var=%s should successed insert into "
                            "op_var2gpu_str, but now failed",
                            op_var));

#ifdef __HIPCC__
      PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(gpu_str_ptr,
                                                iter->first.c_str(),
                                                op_var.length() + 1,
                                                hipMemcpyHostToDevice,
                                                ctx.stream()));
#else
      PADDLE_ENFORCE_GPU_SUCCESS(hipMemcpyAsync(gpu_str_ptr,
                                                 iter->first.c_str(),
                                                 op_var.length() + 1,
                                                 hipMemcpyHostToDevice,
                                                 ctx.stream()));
#endif
    } else {  // get
      auto iter = op_var2gpu_str.find(op_var);
      PADDLE_ENFORCE_EQ(iter != op_var2gpu_str.end(),
                        true,
                        phi::errors::PreconditionNotMet(
                            "op_var=%s should be in the op_var2gpu_str, but "
                            "now can't find it",
                            op_var));
      gpu_str_ptr = reinterpret_cast<char*>(iter->second->ptr());
    }
  }
  return gpu_str_ptr;
}

template <typename T, typename Context>
void CheckNumericsKernel(const Context& ctx,
                         const DenseTensor& tensor,
                         const std::string& op_type,
                         const std::string& var_name) {
  std::call_once(init_multi_gpu_op_var_map_flag, InitMultiGPUOpVarMap);

  int dev_id = tensor.place().device;

  // Print to the standard output.
  char* gpu_str_ptr = GetGpuHintStringPtr<T>(ctx, op_type, var_name, dev_id);

#ifdef __HIPCC__
  // HIP will throw GPU memory access fault if threads > 256
  const size_t threads = 256;
#else
  const size_t threads = 1024;
#endif
  size_t blocks =
      std::min(static_cast<size_t>(128),
               static_cast<size_t>((tensor.numel() + threads - 1) / threads));
#ifdef __HIPCC__
  int print_num = 3;

  hipLaunchKernelGGL(CheckNanInfKernel,
                     dim3(blocks),
                     dim3(threads),
                     0,
                     ctx.stream(),
                     tensor.data<T>(),
                     tensor.numel(),
                     print_num,
                     gpu_str_ptr);
#else
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;

  int64_t numel_max_min = blocks;

  phi::DenseTensor block_num_nan_inf_zero;
  block_num_nan_inf_zero.Resize({static_cast<int64_t>(3 * numel_max_min)});
  int64_t* block_num_nan_ptr =
      ctx.template Alloc<int64_t>(&block_num_nan_inf_zero);
  int64_t* block_num_inf_ptr = block_num_nan_ptr + numel_max_min;
  int64_t* block_num_zero_ptr = block_num_inf_ptr + numel_max_min;

  phi::DenseTensor tensor_block_max_min;
  tensor_block_max_min.Resize({static_cast<int64_t>(3 * numel_max_min)});
  MT* tensor_block_max_ptr = ctx.template Alloc<MT>(&tensor_block_max_min);
  MT* tensor_block_min_ptr = tensor_block_max_ptr + numel_max_min;
  MT* tensor_block_mean_ptr = tensor_block_max_ptr + 2 * numel_max_min;

  FindNanInfAndBlockMaxMin<T, MT>
      <<<blocks, threads, 0, ctx.stream()>>>(tensor.data<T>(),
                                             tensor.numel(),
                                             block_num_nan_ptr,
                                             block_num_inf_ptr,
                                             block_num_zero_ptr,
                                             tensor_block_max_ptr,
                                             tensor_block_min_ptr,
                                             tensor_block_mean_ptr);

  int check_nan_inf_level = FLAGS_check_nan_inf_level;
  FindGlobalMaxMinAndPrint<T, MT>
      <<<1, 1, 0, ctx.stream()>>>(block_num_nan_ptr,
                                  block_num_inf_ptr,
                                  block_num_zero_ptr,
                                  tensor_block_max_ptr,
                                  tensor_block_min_ptr,
                                  tensor_block_mean_ptr,
                                  gpu_str_ptr,
                                  tensor.numel(),
                                  numel_max_min,
                                  check_nan_inf_level);
#endif
}

}  // namespace phi

PD_REGISTER_KERNEL(check_numerics,
                   GPU,
                   ALL_LAYOUT,
                   phi::CheckNumericsKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   phi::dtype::complex<float>,
                   phi::dtype::complex<double>) {}
