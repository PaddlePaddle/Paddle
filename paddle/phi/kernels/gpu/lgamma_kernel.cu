#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/lgamma_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/elementwise_base.h"

namespace phi {
template <typename T>
struct CudaLgammaFunctor {
  __device__ __forceinline__ T operator()(const T x) const {
    return Eigen::numext::lgamma(x);
  }
};
template <typename T, typename Context>
void LgammaKernel(const Context& dev_ctx,
                  const DenseTensor& x,
                  DenseTensor* out) {
  // XKTODO( add gpu kernel implementation. )
  dev_ctx.template Alloc<T>(out);
  std::vector<const DenseTensor*> ins = {&x};
  std::vector<DenseTensor*> outs = {out};
  auto functor = CudaLgammaFunctor<T>();
  phi::funcs::ElementwiseKernel<T>(dev_ctx, ins, &outs, functor);
}
}  // namespace phi

PD_REGISTER_KERNEL(lgamma, GPU, ALL_LAYOUT, phi::LgammaKernel, float, double) {}
