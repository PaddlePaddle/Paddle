#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/triangular_solve_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/expand_kernel.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/common_shape.h"

// See Note [ Why still include the fluid headers? ]
#include "paddle/fluid/memory/allocation/allocator.h"
#include "paddle/fluid/memory/memory.h"

namespace phi {

template <typename T, typename Context>
void TriangularSolveKernel(const Context& dev_ctx,
                           const DenseTensor& x,
                           const DenseTensor& y,
                           bool upper,
                           bool transpose,
                           bool unitriangular,
                           DenseTensor* out) {
  // get broadcast dim
  std::vector<int64_t> x_bst_dims_vec;
  std::vector<int64_t> y_bst_dims_vec;
  std::tie(x_bst_dims_vec, y_bst_dims_vec) =
      funcs::MatrixGetBroadcastDims(x, y);
  int x_bst_ndim = x_bst_dims_vec.size();
  int y_bst_ndim = y_bst_dims_vec.size();

  // Tensor broadcast to 'out' and temp 'x_bst'
  IntArray x_bst_dims(x_bst_dims_vec);
  DenseTensor x_bst = phi::Empty<T, Context>(dev_ctx, x_bst_dims);
  const T* x_bst_data = x_bst.data<T>();
  ExpandKernel<T, Context>(dev_ctx, x, x_bst_dims, &x_bst);

  out->Resize(phi::make_ddim(y_bst_dims_vec));
  T* out_data = dev_ctx.template Alloc<T>(out);
  IntArray y_bst_dims(y_bst_dims_vec);
  ExpandKernel<T, Context>(dev_ctx, y, y_bst_dims, out);

  // calculate use cublas library
  CBLAS_UPLO uplo = upper ? CblasUpper : CblasLower;
  CBLAS_TRANSPOSE transA = transpose ? CblasTrans : CblasNoTrans;
  CBLAS_DIAG diag = unitriangular ? CblasUnit : CblasNonUnit;

  int M = static_cast<int>(y_bst_dims_vec[y_bst_ndim - 2]);
  int N = static_cast<int>(y_bst_dims_vec[y_bst_ndim - 1]);
  auto lda = std::max(1, M);
  auto ldb = std::max(1, N);

  int batch_size = 1;
  for (int i = 0; i < x_bst_ndim - 2; i++) {
    batch_size *= x_bst_dims_vec[i];
  }

  auto blas = phi::funcs::GetBlas<GPUContext, T>(dev_ctx);
  if (batch_size <= 8 && M >= 64) {
    for (auto i = 0; i < batch_size; i++) {
      blas.TRSM(CblasLeft,
                uplo,
                transA,
                diag,
                M,
                N,
                T(1),
                x_bst_data + i * M * M,
                lda,
                out_data + i * N * M,
                ldb);
    }
  } else {
    std::vector<const T*> cpu_ptrs(batch_size * 2);
    for (int i = 0; i < batch_size; ++i) {
      cpu_ptrs[i] = x_bst_data + i * M * M;
      cpu_ptrs[i + batch_size] = out_data + i * M * N;
    }

    // Copy the addresses of A and tmp_b from host to device.
    paddle::memory::allocation::AllocationPtr tmp_gpu_ptrs_data =
        paddle::memory::Alloc(
            dev_ctx.GetPlace(),
            cpu_ptrs.size() * sizeof(T*),
            phi::Stream(reinterpret_cast<phi::StreamId>(dev_ctx.stream())));

    paddle::memory::Copy(dev_ctx.GetPlace(),
                         tmp_gpu_ptrs_data->ptr(),
                         paddle::platform::CPUPlace(),
                         static_cast<void*>(cpu_ptrs.data()),
                         cpu_ptrs.size() * sizeof(T*),
                         dev_ctx.stream());

    const T** gpu_a_ptrs =
        reinterpret_cast<const T**>(tmp_gpu_ptrs_data->ptr());
    T** gpu_b_ptrs =
        reinterpret_cast<T**>(tmp_gpu_ptrs_data->ptr()) + batch_size;
    blas.BatchedTRSM(CblasLeft,
                     uplo,
                     transA,
                     diag,
                     M,
                     N,
                     static_cast<T>(1.0),
                     gpu_a_ptrs,
                     lda,
                     gpu_b_ptrs,
                     ldb,
                     batch_size);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(triangular_solve,
                   GPU,
                   ALL_LAYOUT,
                   phi::TriangularSolveKernel,
                   float,
                   double) {}
