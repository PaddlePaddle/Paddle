#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cross_kernel.h"
#include "paddle/phi/kernels/funcs/reduce_function.h"

namespace phi {

using funcs::IndexCalculator;

template <typename T>
__global__ void Cross(const T* x,
                      const T* y,
                      T* out,
                      const int stride,
                      const int N,
                      IndexCalculator index_calculator) {
  CUDA_KERNEL_LOOP(i, N) {
    int offset = index_calculator(i);

    auto pos0 = offset + 0 * stride;
    auto pos1 = offset + 1 * stride;
    auto pos2 = offset + 2 * stride;

    out[pos0] = x[pos1] * y[pos2] - x[pos2] * y[pos1];
    out[pos1] = x[pos2] * y[pos0] - x[pos0] * y[pos2];
    out[pos2] = x[pos0] * y[pos1] - x[pos1] * y[pos0];
  }
}

template <typename T, typename Context>
void CrossKernel(const Context& dev_ctx,
                 const DenseTensor& x,
                 const DenseTensor& y,
                 int axis,
                 DenseTensor* out) {
  auto& input_x = x;
  auto& input_y = y;
  auto* output = out;
  int dim = axis;

  auto input_x_dims = input_x.dims();
  if (dim != DDim::kMaxRank) {
    PADDLE_ENFORCE_EQ(
        dim < input_x_dims.size() && dim >= (0 - input_x_dims.size()),
        true,
        phi::errors::OutOfRange(
            "Attr(dim) is out of range, It's expected "
            "to be in range of [-%d, %d]. But received Attr(dim) = %d.",
            input_x_dims.size(),
            input_x_dims.size() - 1,
            dim));
    if (dim < 0) {
      dim += input_x_dims.size();
    }

    PADDLE_ENFORCE_EQ(
        input_x_dims[dim] == 3,
        true,
        phi::errors::InvalidArgument(
            "Input(X/Y).dims[dim] must be equal to 3. But received: "
            "Input(X/Y).dims[dim] = [%d].",
            input_x_dims[dim]));
  } else {
    for (auto i = 0; i < input_x_dims.size(); i++) {
      if (input_x_dims[i] == 3) {
        dim = i;
        break;
      }
    }
    PADDLE_ENFORCE_EQ(dim == DDim::kMaxRank,
                      false,
                      phi::errors::InvalidArgument(
                          "There must be at least one dimension 'd' so that "
                          "Input(X/Y).dims()[d] is equal to 3. "
                          "But received: Input(X/Y).dims() == [%s].",
                          input_x_dims));
  }

  std::vector<int> cal_dims;
  std::vector<int> left_strides;
  std::vector<int> full_strides;

  int dims0 = 1;
  int dims1 = 1;
  for (auto i = 0; i < input_x_dims.size(); i++) {
    full_strides.insert(full_strides.begin(), dims0);
    dims0 *= input_x_dims[input_x_dims.size() - i - 1];
    if (i == dim) {
      continue;
    }
    cal_dims.push_back(i);
    left_strides.insert(left_strides.begin(), dims1);
    dims1 *= input_x_dims[input_x_dims.size() - i - 1];
  }

  const auto* input_x_data = input_x.data<T>();
  const auto* input_y_data = input_y.data<T>();

  auto* out_data = dev_ctx.template Alloc<T>(out);

  auto index_calculator = IndexCalculator(
      input_x_dims.size() - 1, cal_dims, left_strides, full_strides);

  int64_t numel = x.numel();

  backends::gpu::GpuLaunchConfig config =
      backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel / 3);

  Cross<<<config.block_per_grid,
          config.thread_per_block,
          0,
          dev_ctx.stream()>>>(input_x_data,
                              input_y_data,
                              out_data,
                              full_strides[dim],
                              numel / 3,
                              index_calculator);
}
}  // namespace phi

PD_REGISTER_KERNEL(
    cross, GPU, ALL_LAYOUT, phi::CrossKernel, float, double, int, int64_t) {}
