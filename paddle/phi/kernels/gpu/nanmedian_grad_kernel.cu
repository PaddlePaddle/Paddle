#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/nanmedian_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_meta.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/impl/nanmedian_grad_kernel_impl.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;
inline int GET_BLOCKS(const int N) {
  return (N + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS;
}

template <typename T>
__global__ void KernelNanmedianGrad(const T* x_ptr,
                                    const int64_t* medians_ptr,
                                    const T* out_grad_ptr,
                                    T* x_grad_ptr,
                                    int64_t stride,
                                    int64_t pre_dim,
                                    T div_factor) {
  CUDA_KERNEL_LOOP(index, pre_dim) {
    int64_t offset = index * stride;
    if (medians_ptr[2 * index] >= 0) {
      if (medians_ptr[2 * index] == medians_ptr[2 * index + 1]) {
        x_grad_ptr[offset + medians_ptr[2 * index]] = out_grad_ptr[index];
      } else {
        x_grad_ptr[offset + medians_ptr[2 * index]] =
            out_grad_ptr[index] / div_factor;
        x_grad_ptr[offset + medians_ptr[2 * index + 1]] =
            out_grad_ptr[index] / div_factor;
      }
    }
  }
}

template <typename T, typename Context>
void CalcMedianGradKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& median_index,
                          const DenseTensor& out_grad,
                          DenseTensor* x_grad,
                          T* x_grad_ptr) {
  phi::funcs::SetConstant<Context, T> set_zero;
  set_zero(dev_ctx, x_grad, static_cast<T>(0));

  auto stream = dev_ctx.stream();
  const T* x_ptr = x.data<T>();
  const int64_t* m_ptr = median_index.data<int64_t>();
  const T* out_grad_ptr = out_grad.data<T>();

  int64_t numel = x.numel();
  auto x_dim = x.dims();
  int64_t x_rank = x_dim.size();
  int64_t stride = x_dim[x_rank - 1];
  int64_t pre_dim = numel / stride;

  T div_factor = static_cast<T>(2.0);
  KernelNanmedianGrad<T>
      <<<GET_BLOCKS(pre_dim), PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          x_ptr, m_ptr, out_grad_ptr, x_grad_ptr, stride, pre_dim, div_factor);
}

template <typename T, typename Context>
void BaseMedianGradKernel(const Context& dev_ctx,
                          const DenseTensor& x,
                          const DenseTensor& median_index,
                          const DenseTensor& out_grad,
                          const IntArray& axes,
                          DenseTensor* x_grad) {
  auto rank = x.dims().size();
  T* x_grad_ptr = dev_ctx.template Alloc<T>(x_grad);
  if (axes.size() && (rank > 1)) {
    DenseTensor tmp_x_grad(*x_grad);
    CalcMedianGradKernel<T, Context>(
        dev_ctx, x, median_index, out_grad, &tmp_x_grad, x_grad_ptr);
    PostprocessMedianGradKernel<T, Context>(dev_ctx, &tmp_x_grad, axes, x_grad);
  } else {
    CalcMedianGradKernel<T, Context>(
        dev_ctx, x, median_index, out_grad, x_grad, x_grad_ptr);
  }
}

template <typename T, typename Context>
void NanmedianGradKernel(const Context& dev_ctx,
                         const DenseTensor& input,
                         const DenseTensor& median_index,
                         const DenseTensor& out_grad,
                         const IntArray& axes,
                         bool keep_dim,
                         DenseTensor* x_grad) {
  BaseMedianGradKernel<T, Context>(
      dev_ctx, input, median_index, out_grad, axes, x_grad);
}

}  // namespace phi

PD_REGISTER_KERNEL(nanmedian_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::NanmedianGradKernel,
                   float,
                   double,
                   int,
                   int64_t,
                   phi::dtype::float16) {}
