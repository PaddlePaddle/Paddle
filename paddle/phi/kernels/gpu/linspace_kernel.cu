#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/linspace_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_primitives.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

template <typename T>
__global__ void LinspaceKernelInner(
    T start, T stop, double step, int64_t size, T* out) {
  int64_t index = blockIdx.x * blockDim.x + threadIdx.x;

  for (; index < size; index += blockDim.x * gridDim.x) {
    if (index < size / 2) {
      out[index] = static_cast<T>(start + step * index);
    } else {
      out[index] = static_cast<T>(stop - step * (size - index - 1));
    }
  }
}

template <typename T>
__global__ void LinspaceSpecialKernel(T start, T* out) {
  out[0] = static_cast<T>(start);
}

template <typename T, typename Context>
void LinspaceKernel(const Context& ctx,
                    const Scalar& start,
                    const Scalar& stop,
                    const Scalar& number,
                    DataType dtype,
                    DenseTensor* out) {
  T start_value = start.to<T>();
  T stop_value = stop.to<T>();
  int64_t num = number.to<int64_t>();
  PADDLE_ENFORCE_GT(
      num,
      0,
      phi::errors::InvalidArgument("The num of linspace op should be larger "
                                   "than 0, but received num is %d",
                                   num));

  out->Resize(phi::make_ddim({num}));
  T* out_data = ctx.template Alloc<T>(out);

  auto stream = ctx.stream();
  if (num != 1) {
    int block = 512;
    int grid = (num + block - 1) / block;
    double step = (static_cast<double>(stop_value - start_value)) / (num - 1);
    LinspaceKernelInner<T><<<grid, block, 0, stream>>>(
        start_value, stop_value, step, num, out_data);
  } else {
    LinspaceSpecialKernel<T><<<1, 1, 0, stream>>>(start_value, out_data);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(linspace,
                   GPU,
                   ALL_LAYOUT,
                   phi::LinspaceKernel,
                   float,
                   int32_t,
                   int64_t,
                   double) {
  kernel->InputAt(0).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(1).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
}
