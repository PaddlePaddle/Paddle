#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/linspace_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"

namespace phi {

template <typename T>
__global__ void LinspaceKernelInner(
    T start, T stop, double step, int64_t size, T* out) {
  int64_t index = blockIdx.x * blockDim.x + threadIdx.x;

  for (; index < size; index += blockDim.x * gridDim.x) {
    if (index < size / 2) {
      out[index] = static_cast<T>(start + step * index);
    } else {
      out[index] = static_cast<T>(stop - step * (size - index - 1));
    }
  }
}

template <typename T>
__global__ void LinspaceSpecialKernel(T start, T* out) {
  out[0] = static_cast<T>(start);
}

template <typename T, typename Context>
T GetValue(const Context& ctx, const DenseTensor& x) {
  T value = static_cast<T>(0);
  if (x.place() != CPUPlace()) {
    DenseTensor cpu_x;
    Copy(ctx, x, CPUPlace(), true, &cpu_x);
    value = cpu_x.data<T>()[0];
  } else {
    value = x.data<T>()[0];
  }
  return value;
}

template <typename T, typename Context>
T GetValueOfExpectedType(const Context& ctx, const DenseTensor& x) {
  switch (x.dtype()) {
    case DataType::FLOAT32:
      return static_cast<T>(GetValue<float, Context>(ctx, x));
    case DataType::FLOAT64:
      return static_cast<T>(GetValue<double, Context>(ctx, x));
    case DataType::INT32:
      return static_cast<T>(GetValue<int32_t, Context>(ctx, x));
    case DataType::INT64:
      return static_cast<T>(GetValue<int64_t, Context>(ctx, x));
    case DataType::FLOAT16:
      return static_cast<T>(GetValue<phi::dtype::float16, Context>(ctx, x));
    case DataType::BFLOAT16:
      return static_cast<T>(GetValue<phi::dtype::bfloat16, Context>(ctx, x));
    case DataType::BOOL:
      return static_cast<T>(GetValue<bool, Context>(ctx, x));
    case DataType::INT16:
      return static_cast<T>(GetValue<int16_t, Context>(ctx, x));
    case DataType::UINT8:
      return static_cast<T>(GetValue<uint8_t, Context>(ctx, x));
    default:
      PADDLE_THROW(phi::errors::Unimplemented(
          "Data type (%s) is not supported when casting data type.",
          x.dtype()));
  }
}

template <typename T, typename Context>
void LinspaceKernel(const Context& ctx,
                    const DenseTensor& start,
                    const DenseTensor& stop,
                    const DenseTensor& number,
                    DataType dtype,
                    DenseTensor* out) {
  T start_value = GetValueOfExpectedType<T, Context>(ctx, start);
  T stop_value = GetValueOfExpectedType<T, Context>(ctx, stop);
  int64_t num = GetValueOfExpectedType<int64_t, Context>(ctx, number);

  PADDLE_ENFORCE_GT(
      num,
      0,
      phi::errors::InvalidArgument("The num of linspace op should be larger "
                                   "than 0, but received num is %d",
                                   num));

  out->Resize(phi::make_ddim({num}));
  T* out_data = ctx.template Alloc<T>(out);

  auto stream = ctx.stream();
  if (num != 1) {
    int block = 512;
    int grid = (num + block - 1) / block;
    double step = (static_cast<double>(stop_value - start_value)) / (num - 1);
    LinspaceKernelInner<T><<<grid, block, 0, stream>>>(
        start_value, stop_value, step, num, out_data);
  } else {
    LinspaceSpecialKernel<T><<<1, 1, 0, stream>>>(start_value, out_data);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(linspace,
                   GPU,
                   ALL_LAYOUT,
                   phi::LinspaceKernel,
                   float,
                   int32_t,
                   int64_t,
                   double) {
  kernel->InputAt(0).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(1).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(2).SetBackend(phi::Backend::ALL_BACKEND);
}
