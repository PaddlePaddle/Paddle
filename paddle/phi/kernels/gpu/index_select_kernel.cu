#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/index_select_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/utils/data_type.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename T, typename IndexT>
__global__ void index_select_cuda_kernel(const T* input,
                                         T* output,
                                         const IndexT* index,
                                         int64_t N,
                                         int64_t stride,
                                         int64_t size,
                                         int64_t delta) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t pre_idx = idx / (stride * size);
  int64_t dim_idx = idx % (stride * size) / stride;
  IndexT src_dim_idx = index[dim_idx];
  int64_t input_idx = idx + (delta * pre_idx + src_dim_idx - dim_idx) * stride;
  output[idx] = input[input_idx];
}

template <typename T, typename Context>
void IndexSelectKernel(const Context& ctx,
                       const DenseTensor& x,
                       const DenseTensor& index,
                       int dim,
                       DenseTensor* output) {
  auto input_dim = x.dims();
  auto output_dim = output->dims();
  dim = dim >= 0 ? dim : dim + input_dim.size();
  auto stride_dim = phi::stride(input_dim);
  int64_t stride = stride_dim[dim];
  int64_t size = output_dim[dim];
  int64_t delta = input_dim[dim] - size;
  const auto& index_type = index.dtype();

  bool index_type_match =
      index_type == phi::DataType::INT64 || index_type == phi::DataType::INT32;
  PADDLE_ENFORCE_EQ(index_type_match,
                    true,
                    phi::errors::InvalidArgument(
                        "Input(Index) holds the wrong type, it holds %s, but "
                        "desires to be %s or %s",
                        index_type,
                        phi::DataType::INT32,
                        phi::DataType::INT64));

  auto* in_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(output);

  int64_t numel = output->numel();
  auto stream = ctx.stream();

  if (index_type == phi::DataType::INT64) {
    const int64_t* index_data = index.data<int64_t>();
    index_select_cuda_kernel<T, int64_t><<<
        (numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
        PADDLE_CUDA_NUM_THREADS,
        0,
        stream>>>(in_data, out_data, index_data, numel, stride, size, delta);
  } else {
    const int* index_data = index.data<int>();
    index_select_cuda_kernel<
        T,
        int><<<(numel + PADDLE_CUDA_NUM_THREADS - 1) / PADDLE_CUDA_NUM_THREADS,
               PADDLE_CUDA_NUM_THREADS,
               0,
               stream>>>(
        in_data, out_data, index_data, numel, stride, size, delta);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(index_select,
                   GPU,
                   ALL_LAYOUT,
                   phi::IndexSelectKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int,
                   int64_t) {}
