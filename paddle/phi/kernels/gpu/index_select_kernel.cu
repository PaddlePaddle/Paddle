// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/index_select_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/utils/data_type.h"
#include "paddle/phi/kernels/gpu/index_select_impl.h"

namespace phi {

using paddle::platform::PADDLE_CUDA_NUM_THREADS;

template <typename T, typename Context>
void IndexSelectKernel(const Context& ctx,
                       const DenseTensor& x,
                       const DenseTensor& index,
                       int dim,
                       DenseTensor* output) {
  auto input_dim = x.dims();
  auto output_dim = output->dims();
  dim = dim >= 0 ? dim : dim + input_dim.size();
  auto stride_dim = phi::stride(input_dim);
  int64_t stride = stride_dim[dim];
  int64_t size = output_dim[dim];
  int64_t delta = input_dim[dim] - size;
  const auto& index_type = index.dtype();

  bool index_type_match =
      index_type == phi::DataType::INT64 || index_type == phi::DataType::INT32;
  PADDLE_ENFORCE_EQ(index_type_match,
                    true,
                    phi::errors::InvalidArgument(
                        "Input(Index) holds the wrong type, it holds %s, but "
                        "desires to be %s or %s",
                        index_type,
                        phi::DataType::INT32,
                        phi::DataType::INT64));

  auto* in_data = x.data<T>();
  T* out_data = ctx.template Alloc<T>(output);

  int64_t numel = output->numel();
  if (numel == 0) {
    return;
  }
  auto stream = ctx.stream();

  unsigned int block_dim = PADDLE_CUDA_NUM_THREADS;
  dim3 grid_dim = dim3((numel + block_dim - 1) / block_dim);
  paddle::platform::LimitGridDim(ctx, &grid_dim);

  if (index_type == phi::DataType::INT64) {
    const int64_t* index_data = index.data<int64_t>();
    index_select_cuda_kernel<T, int64_t><<<grid_dim, block_dim, 0, stream>>>(
        in_data, out_data, index_data, numel, stride, size, delta);
  } else {
    const int* index_data = index.data<int>();
    index_select_cuda_kernel<T, int><<<grid_dim, block_dim, 0, stream>>>(
        in_data, out_data, index_data, numel, stride, size, delta);
  }
}

}  // namespace phi

PD_REGISTER_KERNEL(index_select,
                   GPU,
                   ALL_LAYOUT,
                   phi::IndexSelectKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   int,
                   int64_t) {}
