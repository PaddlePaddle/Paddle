#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include "paddle/phi/backends/gpu/gpu_helper.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/common/pstring.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/strings/strings_serialize_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_launch_config.h"

using pstring = ::phi::dtype::pstring;

namespace phi {
namespace strings {

__global__ void SerializeStringsData(const pstring* src_str,
                                     uint8_t* strings_data,
                                     int32_t* strings_offset,
                                     int64_t numel,
                                     int32_t start_offset) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    strings_offset[0] = start_offset;
    for (int64_t i = 1; i <= numel; ++i) {
      strings_offset[i] = strings_offset[i - 1] + src_str[i - 1].length() + 1;
    }
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(i, numel) {
    memcpy(strings_data + strings_offset[i],
           src_str[i].data(),
           src_str[i].length() + 1);
  }
}

__global__ void SumStringsLen(const pstring* src_ptr, int64_t numel, int* num) {
  extern __shared__ int counter[];
  int thread_counter = 0;
  CUDA_KERNEL_LOOP(i, numel) { thread_counter += src_ptr[i].length() + 1; }
  counter[threadIdx.x] = thread_counter;
  __syncthreads();
  if (threadIdx.x == 0) {
    int block_counter = 0;
    for (int i = 0; i < blockDim.x; ++i) {
      block_counter += counter[i];
    }
    atomicAdd(num, block_counter);
  }
}

template <typename Context>
int GetAllStringsSize(const Context& dev_ctx,
                      const pstring* src_ptr,
                      size_t numel) {
  auto nums_meta =
      phi::DenseTensorMeta(DataType::INT32, {1}, phi::DataLayout::NCHW);
  DenseTensor nums_tensor = phi::Empty(dev_ctx, std::move(nums_meta));
  const auto place = dev_ctx.GetPlace();
  int* nums_ptr = nums_tensor.mutable_data<int>(place);
  phi::backends::gpu::GpuMemsetAsync(
      nums_ptr, 0, sizeof(int), dev_ctx.stream());

  dim3 block_size = dim3(PREDEFINED_BLOCK_SIZE, 1);
  dim3 grid_size =
      dim3((numel + PREDEFINED_BLOCK_SIZE - 1) / PREDEFINED_BLOCK_SIZE, 1);
  SumStringsLen<<<grid_size,
                  block_size,
                  PREDEFINED_BLOCK_SIZE * sizeof(int),
                  dev_ctx.stream()>>>(src_ptr, numel, nums_ptr);
  int num = -1;
#ifdef PADDLE_WITH_HIP
  phi::backends::gpu::GpuMemcpyAsync(
      &num, nums_ptr, sizeof(int), hipMemcpyDeviceToHost, dev_ctx.stream());
#else
  phi::backends::gpu::GpuMemcpyAsync(
      &num, nums_ptr, sizeof(int), hipMemcpyDeviceToHost, dev_ctx.stream());
#endif
  return num;
}

template <typename Context>
void Serialize(const Context& dev_ctx,
               const StringTensor& src,
               DenseTensor* dst) {
  int64_t numel = src.numel();
  auto* src_str = src.data();
  // 1.get the number of bytes of all strings in string tensor
  auto strings_size = GetAllStringsSize(dev_ctx, src_str, numel);
  strings_size += sizeof(int32_t) * (numel + 1);
  dst->ResizeAndAllocate({strings_size});
  auto* strings_data = dst->mutable_data<uint8_t>(src.place());
  auto* strings_offset = reinterpret_cast<int32_t*>(strings_data);
  int32_t start_offset = sizeof(int32_t) * (numel + 1);
  // 2. serialize strings data to dense tensor
  dim3 block_size = dim3(PREDEFINED_BLOCK_SIZE, 1);
  dim3 grid_size =
      dim3((numel + PREDEFINED_BLOCK_SIZE - 1) / PREDEFINED_BLOCK_SIZE, 1);

  SerializeStringsData<<<grid_size, block_size, 0, dev_ctx.stream()>>>(
      src_str, strings_data, strings_offset, numel, start_offset);
}

}  // namespace strings
}  // namespace phi

PD_REGISTER_GENERAL_KERNEL(strings_serialize,
                           GPU,
                           ALL_LAYOUT,
                           phi::strings::Serialize<phi::GPUContext>,
                           pstring) {}
