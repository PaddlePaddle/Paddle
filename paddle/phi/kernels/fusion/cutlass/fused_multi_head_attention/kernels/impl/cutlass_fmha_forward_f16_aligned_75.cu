#include "hip/hip_runtime.h"
// This file is auto-generated. See "generate_kernels.py"
#ifndef XFORMERS_MEM_EFF_ATTENTION_DISABLE_FORWARD
#include "../../kernel_forward.h"

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, false>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, false>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, false>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, false>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, false>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, true>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, true>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, true>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, false, true>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, false>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, false>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, false>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, false>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, false>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, true>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, true>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, true>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, true, true, true>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, false>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, false>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, false>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, false>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, false>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, true>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, true>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, true>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, false, true>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, false>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, false>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, false>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, false>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, false>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, true>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, true>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, true>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, true, true, true>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, false>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, false>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, false>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, false>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, false>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, true>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, true>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, true>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, false, true>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, false>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, false>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, false>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, false>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, false>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void __launch_bounds__(
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, true>::kNumThreads,
    AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, true>::kMinBlocksPerSm)
attention_kernel_batched<AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, true>>(typename AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, true>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750 && __CUDA_ARCH__ < 800
  if (!params.advance_to_block()) {
    return;
  }
  AttentionKernel<cutlass::half_t, cutlass::arch::Sm75, true, 32, 128, false, true, true>::attention_kernel(params);
  return;
#endif
    printf(
        "FATAL: kernel `attention_kernel_batched` is for sm75-sm80, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
#endif // XFORMERS_MEM_EFF_ATTENTION_DISABLE_FORWARD
