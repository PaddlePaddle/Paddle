// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#pragma once
#include "cutlass/conv/kernel/default_conv2d_fprop_with_broadcast.h"
#include "cutlass/epilogue/thread/linear_combination_residual_block.h"
#include "paddle/phi/kernels/fusion/cutlass/conv2d/conv2d_util.h"

namespace phi {
namespace fusion {

namespace cutlass_internal {

template <typename TShape, typename WShape, int Alignment = 8>
cutlass::Status Conv2dBiasAddReluImpl(ConvAllParams params) {
  using EpilogueOp = cutlass::epilogue::thread::LinearCombinationResidualBlock<
      cutlass::half_t,
      float,
      float,
      cutlass::half_t,
      Alignment,
      cutlass::epilogue::thread::Identity,
      cutlass::plus,
      cutlass::epilogue::thread::ReLu>;

  using Conv2dFpropKernel =
      typename cutlass::conv::kernel::DefaultConv2dFpropWithBroadcast<
          cutlass::half_t,
          cutlass::layout::TensorNHWC,
          cutlass::half_t,
          cutlass::layout::TensorNHWC,
          cutlass::half_t,
          cutlass::layout::TensorNHWC,
          float,
          cutlass::arch::OpClassTensorOp,
          cutlass::arch::Sm75,
          TShape,
          WShape,
          cutlass::gemm::GemmShape<16, 8, 8>,
          EpilogueOp,
          cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>,
          2,
          cutlass::arch::OpMultiplyAdd,
          cutlass::conv::IteratorAlgorithm::kOptimized,
          cutlass::conv::StrideSupport::kStrided,
          Alignment,
          Alignment>::Kernel;

  using ImplicitGemm =
      cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

  const half *input = params.input;
  const half *weight = params.weight;
  const half *bias = params.bias;

  half *output = params.output;
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int kh = params.kh;
  int kw = params.kw;
  int oc = params.oc;
  int pad_h0 = params.pad_h0;
  int pad_w0 = params.pad_w0;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;
  const half *residual = params.residual;

  int oh = params.oh;
  int ow = params.ow;
  int dilation_h = params.dilation_h;
  int dilation_w = params.dilation_w;

  cutlass::conv::Conv2dProblemSize problem_size(
      {batch, ih, iw, ic},
      {oc, kh, kw, ic},
      {pad_h0, 0, pad_w0, 0},
      {stride_h, stride_w},
      {dilation_h, dilation_w},
      {batch, oh, ow, oc},
      cutlass::conv::Mode::kCrossCorrelation,
      1);

  typename ImplicitGemm::Arguments arguments{
      problem_size,
      {(cutlass::half_t *)input, {ic, ic * iw, ic * iw * ih}},
      {(cutlass::half_t *)weight, {ic, ic * kw, ic * kw * kh}},
      {(cutlass::half_t *)residual, {oc, oc * ow, oc * ow * oh}},
      {(cutlass::half_t *)output, {oc, oc * ow, oc * ow * oh}},
      {1.f, 1.f},
      cutlass::conv::SplitKMode::kSerial,
      (cutlass::half_t *)(bias),
      nullptr,
      0,
      oc};

  ImplicitGemm implicit_gemm_op;
  size_t bytes = implicit_gemm_op.get_workspace_size(arguments);

  auto ctx = params.ctx;
  auto stream = ctx->stream();
  paddle::memory::allocation::AllocationPtr tmp_gpu_ptrs_data =
      paddle::memory::Alloc(
          ctx->GetPlace(),
          bytes,
          phi::Stream(reinterpret_cast<phi::StreamId>(stream)));
  void *workspace = tmp_gpu_ptrs_data->ptr();

  cutlass::Status status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op.initialize(arguments, workspace);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op(stream);
  CUTLASS_CHECK(status);
  return status;
}

// config 0
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 64, 64>,
                          cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 1
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 32, 64>,
                          cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 2
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 32, 64>,
                          cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 3
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 64, 64>,
                          cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 4
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 64, 32>,
                          cutlass::gemm::GemmShape<32, 32, 32>>(ConvAllParams);
// config 5
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 128, 32>,
                          cutlass::gemm::GemmShape<32, 64, 32>>(ConvAllParams);
// config 6
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 128, 64>,
                          cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 7
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 256, 32>,
                          cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 8
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 64, 32>,
                          cutlass::gemm::GemmShape<64, 32, 32>>(ConvAllParams);
// config 9
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 128, 32>,
                          cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 10
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 256, 32>,
                          cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 11
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<256, 64, 32>,
                          cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 12
template cutlass::Status
    Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<256, 128, 32>,
                          cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);

std::vector<std::function<cutlass::Status(ConvAllParams)>>
    conv2d_bias_add_relu_all_func = {
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 64, 32>,
                              cutlass::gemm::GemmShape<32, 32, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 64, 64>,
                              cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 32, 64>,
                              cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 32, 64>,
                              cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 64, 64>,
                              cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 128, 32>,
                              cutlass::gemm::GemmShape<32, 64, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 128, 64>,
                              cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<64, 256, 32>,
                              cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 64, 32>,
                              cutlass::gemm::GemmShape<64, 32, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 128, 32>,
                              cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<128, 256, 32>,
                              cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<256, 64, 32>,
                              cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasAddReluImpl<cutlass::gemm::GemmShape<256, 128, 32>,
                              cutlass::gemm::GemmShape<64, 64, 32>>};
std::map<std::vector<int>, int> map_problem_conv2d_bias_add_relu;

void Conv2dBiasAddRelu(ConvAllParams params) {
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int kh = params.kh;
  int kw = params.kw;
  int oc = params.oc;
  int pad_h0 = params.pad_h0;
  int pad_w0 = params.pad_w0;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;

  std::vector<int> problem_size = {
      batch, ic, ih, iw, kh, kw, oc, pad_h0, pad_w0, stride_h, stride_w};

  if (map_problem_conv2d_bias_add_relu.count(problem_size)) {
    conv2d_bias_add_relu_all_func[map_problem_conv2d_bias_add_relu.at(
        problem_size)](params);
    return;
  }
  // config 6's diff is large.
  conv2d_bias_add_relu_all_func[6] = nullptr;

  int best_config_index = ProfileToGetBestConfig(
      conv2d_bias_add_relu_all_func, params, CONV2D_BIAS_ADD_RELU);
  map_problem_conv2d_bias_add_relu[problem_size] = best_config_index;
  conv2d_bias_add_relu_all_func[best_config_index](params);
}
}  // namespace cutlass_internal
}  // namespace fusion
}  // namespace phi
