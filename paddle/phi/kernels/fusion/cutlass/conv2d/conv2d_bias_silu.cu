#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#pragma once
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/epilogue/thread/linear_combination_silu.h"
#include "paddle/phi/kernels/fusion/cutlass/conv2d/conv2d_all.h"
#include "paddle/phi/kernels/fusion/cutlass/conv2d/conv2d_util.h"

namespace phi {
namespace fusion {

template <typename TShape, typename WShape, int Alignment = 8>
cutlass::Status cutlass_nhwc_conv2d_bias_silu(ConvAllParams params) {
  using ElementAccumulator = float;
  using ElementComputeEpilogue = float;
  using ElementInputA = cutlass::half_t;
  using ElementInputB = cutlass::half_t;
  using ElementOutput = cutlass::half_t;
  using LayoutInputA = cutlass::layout::TensorNHWC;
  using LayoutInputB = cutlass::layout::TensorNHWC;
  using LayoutOutput = cutlass::layout::TensorNHWC;
  using MMAOp = cutlass::arch::OpClassTensorOp;
  using SmArch = cutlass::arch::Sm75;
  using ThreadblockShape = TShape;
  using WarpShape = WShape;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 8>;
  using SwizzleThreadBlock =
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>;
  constexpr int NumStages = 2;
  static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm =
      cutlass::conv::IteratorAlgorithm::kOptimized;
  using EpilogueOp =
      cutlass::epilogue::thread::LinearCombinationSilu<ElementOutput,
                                                       Alignment,
                                                       float,
                                                       ElementComputeEpilogue>;

  using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementAccumulator,
      MMAOp,
      SmArch,
      ThreadblockShape,
      WarpShape,
      InstructionShape,
      EpilogueOp,
      SwizzleThreadBlock,
      NumStages,
      cutlass::arch::OpMultiplyAdd,
      IteratorAlgorithm,
      cutlass::conv::StrideSupport::kStrided,
      Alignment,
      Alignment>::Kernel;
  using ImplicitGemm =
      cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

  const half *input = params.input;
  const half *weight = params.weight;
  const half *bias = params.bias;
  half *output = params.output;
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int kh = params.kh;
  int kw = params.kw;
  int oc = params.oc;
  int pad_h0 = params.pad_h0;
  int pad_w0 = params.pad_w0;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;

  int oh = params.oh;
  int ow = params.ow;
  int dilation_h = params.dilation_h;
  int dilation_w = params.dilation_w;

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;
  cutlass::conv::Conv2dProblemSize problem_size({batch, ih, iw, ic},
                                                {oc, kh, kw, ic},
                                                {pad_h0, 0, pad_w0, 0},
                                                {stride_h, stride_w},
                                                {dilation_h, dilation_w},
                                                {batch, oh, ow, oc},
                                                mode,
                                                1);

  typename ImplicitGemm::Arguments arguments{
      problem_size,
      {(cutlass::half_t *)(input), {ic, ic * iw, ic * iw * ih}},
      {(cutlass::half_t *)(weight), {ic, ic * kw, ic * kw * kh}},
      {(cutlass::half_t *)(bias), {0, 0, 0}},
      {(cutlass::half_t *)(output), {oc, oc * ow, oc * ow * oh}},
      {1.f, 1.f}};

  ImplicitGemm implicit_gemm_op;
  size_t bytes = implicit_gemm_op.get_workspace_size(arguments);
  void *workspace;
  PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(&workspace, bytes));

  cutlass::Status status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op.initialize(arguments, workspace);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op(params.stream);
  CUTLASS_CHECK(status);
  PADDLE_ENFORCE_GPU_SUCCESS(hipFree(workspace));
  return status;
}

// config 0
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 1
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<64, 32, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 2
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<128, 32, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 3
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<128, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 4
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32>>(ConvAllParams);
// config 5
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32>>(ConvAllParams);
// config 6
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 7
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<64, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 8
template cutlass::Status cutlass_nhwc_conv2d_bias_silu<
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32>>(ConvAllParams);

std::vector<std::function<cutlass::Status(ConvAllParams)>>
    cutlass_conv2d_bias_silu_all_func = {
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<64, 64, 64>,
                                      cutlass::gemm::GemmShape<32, 32, 64>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<64, 32, 64>,
                                      cutlass::gemm::GemmShape<32, 32, 64>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<128, 32, 64>,
                                      cutlass::gemm::GemmShape<32, 32, 64>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<128, 64, 64>,
                                      cutlass::gemm::GemmShape<32, 32, 64>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<64, 64, 32>,
                                      cutlass::gemm::GemmShape<32, 32, 32>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<64, 128, 32>,
                                      cutlass::gemm::GemmShape<32, 64, 32>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<64, 128, 64>,
                                      cutlass::gemm::GemmShape<64, 64, 32>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<64, 256, 32>,
                                      cutlass::gemm::GemmShape<64, 64, 32>>,
        cutlass_nhwc_conv2d_bias_silu<cutlass::gemm::GemmShape<128, 64, 32>,
                                      cutlass::gemm::GemmShape<64, 32, 32>>};

std::map<std::vector<int>, int> map_problem_conv2d_bias_silu;

void cutlass_conv2d_bias_silu(ConvAllParams params) {
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int kh = params.kh;
  int kw = params.kw;
  int oc = params.oc;
  int pad_h0 = params.pad_h0;
  int pad_w0 = params.pad_w0;
  int stride_h = params.stride_h;
  int stride_w = params.stride_w;

  std::vector<int> problem_size = {
      batch, ic, ih, iw, kh, kw, oc, pad_h0, pad_w0, stride_h, stride_w};

  if (map_problem_conv2d_bias_silu.count(problem_size)) {
    cutlass_conv2d_bias_silu_all_func[map_problem_conv2d_bias_silu.at(
        problem_size)](params);
    return;
  }

  float min_time = 100000.f;
  for (int i = 0; i < cutlass_conv2d_bias_silu_all_func.size(); i++) {
    cutlass::Status status;
    auto func = cutlass_conv2d_bias_silu_all_func[i];
    for (int ii = 0; ii < WARMUP; ii++) {
      status = func(params);
    }

    hipEvent_t beg, end;
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventCreate(&beg));
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventCreate(&end));
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventRecord(beg));
    for (int ii = 0; ii < REPEAT; ii++) {
      status = func(params);
    }

    PADDLE_ENFORCE_GPU_SUCCESS(hipEventRecord(end));
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventSynchronize(end));
    float elapsed_time;
    PADDLE_ENFORCE_GPU_SUCCESS(hipEventElapsedTime(&elapsed_time, beg, end));
    if (elapsed_time < min_time && status == cutlass::Status::kSuccess) {
      min_time = elapsed_time;
      map_problem_conv2d_bias_silu[problem_size] = i;
    }

    // debug code
    VLOG(3) << "conv2d_bias_silu: tactic " << i << " has max diff "
            << conv2d_diff_gpu(params, CONV2D_BIAS_SILU)
            << " compared with baseline";
  }
  PADDLE_ENFORCE_EQ(
      map_problem_conv2d_bias_silu.count(problem_size),
      true,
      phi::errors::PreconditionNotMet("Can't find any cutlass kernel "
                                      "for this conv2d_bias_silu op."));
}

}  // namespace fusion
}  // namespace phi
