// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#pragma once

#include <iostream>
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/epilogue/thread/linear_combination_bias_relu.h"
#include "paddle/phi/kernels/fusion/cutlass/conv2d/conv2d_util.h"

namespace phi {
namespace fusion {
namespace cutlass_internal {
template <typename TShape, typename WShape, int Alignment = 8>
cutlass::Status Conv2dBiasReluImpl(ConvAllParams params) {
  using ElementAccumulator = float;
  using ElementComputeEpilogue = float;
  using ElementInputA = cutlass::half_t;
  using ElementInputB = cutlass::half_t;
  using ElementOutput = cutlass::half_t;
  using LayoutInputA = cutlass::layout::TensorNHWC;
  using LayoutInputB = cutlass::layout::TensorNHWC;
  using LayoutOutput = cutlass::layout::TensorNHWC;
  using MMAOp = cutlass::arch::OpClassTensorOp;
  using SmArch = cutlass::arch::Sm75;
  using ThreadblockShape = TShape;
  using WarpShape = WShape;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 8>;
  using SwizzleThreadBlock =
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>;
  constexpr int NumStages = 2;
  static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm =
      cutlass::conv::IteratorAlgorithm::kOptimized;
  using EpilogueOp =
      cutlass::epilogue::thread::LinearCombinationRelu<ElementOutput,
                                                       Alignment,
                                                       float,
                                                       ElementComputeEpilogue>;

  using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementAccumulator,
      MMAOp,
      SmArch,
      ThreadblockShape,
      WarpShape,
      InstructionShape,
      EpilogueOp,
      SwizzleThreadBlock,
      NumStages,
      cutlass::arch::OpMultiplyAdd,
      IteratorAlgorithm,
      cutlass::conv::StrideSupport::kStrided,
      Alignment,
      Alignment>::Kernel;
  using ImplicitGemm =
      cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

  const half *input = params.input;
  const half *weight = params.weight;
  const half *bias = params.bias;
  half *output = params.output;
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int kh = params.kh;
  int kw = params.kw;
  int oc = params.oc;
  int pad_h0 = params.pad_h0;
  int pad_w0 = params.pad_w0;

  int stride_h = params.stride_h;
  int stride_w = params.stride_w;
  int oh = params.oh;
  int ow = params.ow;
  int dilation_h = params.dilation_h;
  int dilation_w = params.dilation_w;

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;
  cutlass::conv::Conv2dProblemSize problem_size({batch, ih, iw, ic},
                                                {oc, kh, kw, ic},
                                                {pad_h0, 0, pad_w0, 0},
                                                {stride_h, stride_w},
                                                {dilation_h, dilation_w},
                                                {batch, oh, ow, oc},
                                                mode,
                                                1);

  typename ImplicitGemm::Arguments arguments{
      problem_size,
      {(cutlass::half_t *)(input), {ic, ic * iw, ic * iw * ih}},
      {(cutlass::half_t *)(weight), {ic, ic * kw, ic * kw * kh}},
      {(cutlass::half_t *)(bias), {0, 0, 0}},
      {(cutlass::half_t *)(output), {oc, oc * ow, oc * ow * oh}},
      {1.f, 1.f}};

  ImplicitGemm implicit_gemm_op;
  size_t bytes = implicit_gemm_op.get_workspace_size(arguments);

  auto ctx = params.ctx;
  auto stream = ctx->stream();
  paddle::memory::allocation::AllocationPtr tmp_gpu_ptrs_data =
      paddle::memory::Alloc(
          ctx->GetPlace(),
          bytes,
          phi::Stream(reinterpret_cast<phi::StreamId>(stream)));
  void *workspace = tmp_gpu_ptrs_data->ptr();

  cutlass::Status status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op.initialize(arguments, workspace);
  CUTLASS_CHECK(status);
  status = implicit_gemm_op(stream);
  CUTLASS_CHECK(status);
  return status;
}

// config 0
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 64, 64>,
                       cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 1
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 32, 64>,
                       cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 2
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<128, 32, 64>,
                       cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 3
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<128, 64, 64>,
                       cutlass::gemm::GemmShape<32, 32, 64>>(ConvAllParams);
// config 4
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 64, 32>,
                       cutlass::gemm::GemmShape<32, 32, 32>>(ConvAllParams);
// config 5
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 128, 32>,
                       cutlass::gemm::GemmShape<32, 64, 32>>(ConvAllParams);
// config 6
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 128, 64>,
                       cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 7
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 256, 32>,
                       cutlass::gemm::GemmShape<64, 64, 32>>(ConvAllParams);
// config 8
template cutlass::Status
    Conv2dBiasReluImpl<cutlass::gemm::GemmShape<128, 64, 32>,
                       cutlass::gemm::GemmShape<64, 32, 32>>(ConvAllParams);

std::vector<std::function<cutlass::Status(ConvAllParams)>>
    conv2d_bias_relu_all_func = {
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 64, 64>,
                           cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 32, 64>,
                           cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<128, 32, 64>,
                           cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<128, 64, 64>,
                           cutlass::gemm::GemmShape<32, 32, 64>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 64, 32>,
                           cutlass::gemm::GemmShape<32, 32, 32>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 128, 32>,
                           cutlass::gemm::GemmShape<32, 64, 32>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 128, 64>,
                           cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<64, 256, 32>,
                           cutlass::gemm::GemmShape<64, 64, 32>>,
        Conv2dBiasReluImpl<cutlass::gemm::GemmShape<128, 64, 32>,
                           cutlass::gemm::GemmShape<64, 32, 32>>};
std::map<std::vector<int>, int> map_problem_conv2d_bias_relu;

void Conv2dBiasRelu(ConvAllParams params) {
  int batch = params.batch;
  int ic = params.ic;
  int ih = params.ih;
  int iw = params.iw;
  int kh = params.kh;
  int kw = params.kw;
  int oc = params.oc;
  int pad_h0 = params.pad_h0;
  int pad_w0 = params.pad_w0;

  int stride_h = params.stride_h;
  int stride_w = params.stride_w;

  std::vector<int> problem_size = {
      batch, ic, ih, iw, kh, kw, oc, pad_h0, pad_w0, stride_h, stride_w};

  if (map_problem_conv2d_bias_relu.count(problem_size)) {
    conv2d_bias_relu_all_func[map_problem_conv2d_bias_relu.at(problem_size)](
        params);
    return;
  }

  int best_config_index = ProfileToGetBestConfig(
      conv2d_bias_relu_all_func, params, CONV2D_BIAS_RELU);

  map_problem_conv2d_bias_relu[problem_size] = best_config_index;
  conv2d_bias_relu_all_func[best_config_index](params);
}
}  // namespace cutlass_internal
}  // namespace fusion
}  // namespace phi
