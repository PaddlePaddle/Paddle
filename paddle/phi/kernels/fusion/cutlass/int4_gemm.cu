#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "cutlass/numeric_conversion.h"

#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"

#include "paddle/phi/kernels/fusion/cutlass/int4_gemm/int4_gemm_decl.h"
#include "paddle/phi/kernels/fusion/cutlass/int4_gemm/int4_gemm_util.h"

namespace phi {
namespace fusion {
namespace cutlass_gemm_internal {
template <typename T, typename Context>
void Int4GemmKernel(const Context &ctx,
                    const DenseTensor &x,
                    const DenseTensor &y,
                    const DenseTensor &bias,
                    DenseTensor *out,
                    const bool &trans_x,
                    const bool &trans_y,
                    const std::string &activation) {
  ctx.template Alloc<T>(output);
  auto x_dims = x.dims();
  auto y_dims = y.dims();
  auto bias_dims = bias.dims();
  auto out_dims = out->dims();
  CHECK_EQ(x_dims.size() == 2UL, true);
  CHECK_EQ(y_dims.size() == 2UL, true);
  CHECK_EQ(bias_dims.size() == 1UL, true);

  CHECK_EQ(out_dims.size() == 2UL, true);

  const int m = x_dims[0];
  const int kx = x_dims[1];
  const int ky = y_dims[0];
  const int n = y_dims[1];

  CHECK_EQ(kx, ky);

  int sm = getSMVersion();
  if (sm != 75 && sm != 80) {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Cutlass does not support int4 gemm on sm %d", sm));
  }

  cutlass::Array<T, m *kx> *source_x =
      reinterpret_cast<cutlass::Array<T, m * kx> *>(x.data());
  cutlass::Array<T, kx *n> *source_y =
      reinterpret_cast<cutlass::Array<T, ky * n> *>(y.data());
  cutlass::Array<T, m *n> *source_bias =
      reinterpret_cast<cutlass::Array<T, m * n> *>(bias.data());

  cutlass::NumericArrayConverter<cutlass::int4b_t, T, m * kx> convert_x;
  cutlass::NumericArrayConverter<cutlass::int4b_t, T, ky * n> convert_y;
  cutlass::NumericArrayConverter<cutlass::int4b_t, T, m * n> convert_bias;

  cutlass::Array<cutlass::int4b_t, m * kx> *destination_x;
  *destination_x = convert_x(*source_x);
  cutlass::Array<cutlass::int4b_t, ky * n> *destination_y;
  *destination_x = convert_y(*source_y);
  cutlass::Array<cutlass::int4b_t, m * n> *destination_bias;
  *destination_bias = convert_x(*source_bias);
  cutlass::int4b_t *destination_output;

  GemmAllParams params = {
      reinterpret_cast<const cutlass::int4b_t *>(destination_x->data()),
      reinterpret_cast<const cutlass::int4b_t *>(destination_y->data()),
      reinterpret_cast<const cutlass::int4b_t *>(destination_bias->data()),
      destination_output,
      1,
      m,
      n,
      k,
      &ctx};
  if (activation == "identity") {
    Int4GemmBias(params);
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Cutlass dose not support this activation on int4: %s.",
        activation.c_str()));
  }
  out->set_layout(ALL_LAYOUT);
}
}  // namespace cutlass_gemm_internal
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(int4_gemm_cutlass,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::cutlass_gemm_internal::Int4GemmKernel,
                   int8_t) {}
