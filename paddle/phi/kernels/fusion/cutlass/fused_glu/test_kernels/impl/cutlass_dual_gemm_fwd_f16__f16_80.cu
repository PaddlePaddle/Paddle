#include "hip/hip_runtime.h"
// This file is auto-generated. See "generate_kernels.py"
#include "../../cutlass_dual_gemm.h"

template<>
__global__ void DualKernel<cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::Sigmoid, cutlass::arch::Sm80>>(typename cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::Sigmoid, cutlass::arch::Sm80>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 900
  using Operator = cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::Sigmoid, cutlass::arch::Sm80>; 
  Operator op;
  op(params);
#endif
    printf(
        "FATAL: kernel `DualKernel` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void DualKernel<cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::SiLu, cutlass::arch::Sm80>>(typename cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::SiLu, cutlass::arch::Sm80>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 900
  using Operator = cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::SiLu, cutlass::arch::Sm80>; 
  Operator op;
  op(params);
#endif
    printf(
        "FATAL: kernel `DualKernel` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void DualKernel<cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::GELU_taylor, cutlass::arch::Sm80>>(typename cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::GELU_taylor, cutlass::arch::Sm80>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 900
  using Operator = cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, true, cutlass::epilogue::thread::GELU_taylor, cutlass::arch::Sm80>; 
  Operator op;
  op(params);
#endif
    printf(
        "FATAL: kernel `DualKernel` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void DualKernel<cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::Sigmoid, cutlass::arch::Sm80>>(typename cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::Sigmoid, cutlass::arch::Sm80>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 900
  using Operator = cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::Sigmoid, cutlass::arch::Sm80>; 
  Operator op;
  op(params);
#endif
    printf(
        "FATAL: kernel `DualKernel` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void DualKernel<cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::SiLu, cutlass::arch::Sm80>>(typename cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::SiLu, cutlass::arch::Sm80>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 900
  using Operator = cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::SiLu, cutlass::arch::Sm80>; 
  Operator op;
  op(params);
#endif
    printf(
        "FATAL: kernel `DualKernel` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}

template<>
__global__ void DualKernel<cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::GELU_taylor, cutlass::arch::Sm80>>(typename cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::GELU_taylor, cutlass::arch::Sm80>::Params params) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800 && __CUDA_ARCH__ < 900
  using Operator = cutlass::gemm::kernel::DualGemm<cutlass::half_t, cutlass::half_t, false, cutlass::epilogue::thread::GELU_taylor, cutlass::arch::Sm80>; 
  Operator op;
  op(params);
#endif
    printf(
        "FATAL: kernel `DualKernel` is for sm80-sm90, but was built for sm%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
