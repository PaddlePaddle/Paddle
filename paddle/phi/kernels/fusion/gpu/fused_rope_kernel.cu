#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/fusion/gpu/fused_rope_utils.h"
namespace phi {
namespace fusion {

template <typename T, typename Context>
void FusedRopeKernel(const Context& dev_ctx,
                     const DenseTensor& q,
                     const paddle::optional<DenseTensor>& k,
                     const paddle::optional<DenseTensor>& v,
                     const paddle::optional<DenseTensor>& sin,
                     const paddle::optional<DenseTensor>& cos,
                     bool use_neox_rotary_style,
                     DenseTensor* out_q,
                     DenseTensor* out_k,
                     DenseTensor* out_v) {
  int numel = q.numel();
  if (numel <= 0) return;
  dev_ctx.template Alloc<T>(out_q);
  out_q->Resize(q.dims());
  // small size for broadcast
  auto batch_size = q.dims()[0];
  auto num_heads = q.dims()[2];
  auto head_dim = q.dims()[3];
  auto seq_len = q.dims()[1];
  PADDLE_ENFORCE_NE(head_dim % 2,
                    1,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 2."));

  constexpr const int vec_size = 2;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

  int grid = config.block_per_grid.x;
  int block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;

  ins_data[0] = q.data<T>();
  outs_data[0] = out_q->data<T>();
  int num_inputs = 0;

  if (k.get_ptr()) {
    dev_ctx.template Alloc<T>(out_k);
    out_k->Resize(q.dims());
    ins_data[1] = k->data<T>();
    outs_data[1] = out_k->data<T>();
    num_inputs++;
  }

  if (v.get_ptr()) {
    dev_ctx.template Alloc<T>(out_v);
    out_v->Resize(q.dims());
    ins_data[2] = v->data<T>();
    outs_data[2] = out_v->data<T>();
    num_inputs++;
  }

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType div_c = static_cast<MPType>(1.0f / head_dim);

  bool flag_sin_cos = false;

  if (sin.get_ptr() && cos.get_ptr()) {
    PADDLE_ENFORCE_EQ(sin.get_ptr()->dims(),
                      cos.get_ptr()->dims(),
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be the same."));
    auto sin_dims = sin.get_ptr()->dims();
    int dims_size = sin_dims.size();
    PADDLE_ENFORCE_NE((dims_size == 2 || dims_size == 4),
                      false,
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be 2 or 4."));
    if (dims_size == 4) {
      PADDLE_ENFORCE_NE(
          (sin_dims[0] == 1 && sin_dims[1] == 1),
          false,
          phi::errors::InvalidArgument(
              "The batch_size and num_heads of sin and cos must be 1."));
    }
    PADDLE_ENFORCE_NE(
        (sin_dims[dims_size - 1] == head_dim &&
         sin_dims[dims_size - 2] == seq_len),
        false,
        phi::errors::InvalidArgument("The seq_len and head_dim of sin and cos "
                                     "must be the same as those of q."));

    sin_cos_data[0] = sin->data<T>();
    sin_cos_data[1] = cos->data<T>();

    flag_sin_cos = true;
  }

  int sign = 1;
  VectorizedFusedRopeKernel<T, MPType, vec_size>
      <<<grid, block, 0, stream>>>(ins_data,
                                   sin_cos_data,
                                   flag_sin_cos,
                                   use_neox_rotary_style,
                                   sign,
                                   batch_size,
                                   seq_len,
                                   num_heads,
                                   head_dim,
                                   outs_data,
                                   num_inputs,
                                   div_c);
}
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_rotary_position_embedding,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRopeKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
