#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/fusion/gpu/fused_rope_utils.h"

namespace phi {
namespace fusion {

template <typename T, typename Context>
void FusedRopeKernel(const Context& dev_ctx,
                     const DenseTensor& q,
                     const paddle::optional<DenseTensor>& k,
                     const paddle::optional<DenseTensor>& v,
                     const paddle::optional<DenseTensor>& sin,
                     const paddle::optional<DenseTensor>& cos,
                     const paddle::optional<DenseTensor>& position_ids,
                     bool use_neox_rotary_style,
                     DenseTensor* out_q,
                     DenseTensor* out_k,
                     DenseTensor* out_v) {
  int64_t numel = q.numel();
  if (numel <= 0) return;
  dev_ctx.template Alloc<T>(out_q);

  // q.shape: [batch_size, seq_len, num_heads, head_dim]
  auto batch_size = q.dims()[0];
  auto seq_len = q.dims()[1];
  auto num_heads = q.dims()[2];
  auto head_dim = q.dims()[3];
  PADDLE_ENFORCE_EQ(head_dim % 2,
                    0,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 2."));

  constexpr const int vec_size = 2;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

  int64_t grid = config.block_per_grid.x;
  int64_t block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;
  const int64_t* position_ids_data;

  ins_data[0] = q.data<T>();
  outs_data[0] = out_q->data<T>();
  int num_inputs = 0;

  if (k.get_ptr()) {
    dev_ctx.template Alloc<T>(out_k);
    ins_data[1] = k->data<T>();
    outs_data[1] = out_k->data<T>();
    num_inputs++;
  }

  if (v.get_ptr()) {
    dev_ctx.template Alloc<T>(out_v);
    ins_data[2] = v->data<T>();
    outs_data[2] = out_v->data<T>();
    num_inputs++;
  }

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType div_c = static_cast<MPType>(1.0f / head_dim);

  bool flag_sin_cos = false;

  if (sin.get_ptr() && cos.get_ptr()) {
    PADDLE_ENFORCE_EQ(sin.get_ptr()->dims(),
                      cos.get_ptr()->dims(),
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be the same. But "
                          "recieved sin's dims is {%s}, cos's dims is {%s}.",
                          sin.get_ptr()->dims(),
                          cos.get_ptr()->dims()));

    auto sin_dims = sin.get_ptr()->dims();
    int dims_size = sin_dims.size();
    PADDLE_ENFORCE_EQ(
        (dims_size == 2 || dims_size == 4),
        true,
        phi::errors::InvalidArgument("The dims of sin and cos is expected to "
                                     "be 2 or 4, but recieved %d.",
                                     dims_size));
    if (dims_size == 4) {
      // sin.shape: [1, seq_len, 1, head_dim]
      PADDLE_ENFORCE_EQ(
          (sin_dims[0] == 1 && sin_dims[2] == 1),
          true,
          phi::errors::InvalidArgument(
              "The batch_size and num_heads of sin and cos must be 1."));
    }
    int sin_seq_len_dim = (dims_size) == 4 ? 1 : 0;
    PADDLE_ENFORCE_EQ((sin_dims[dims_size - 1] == head_dim &&
                       sin_dims[sin_seq_len_dim] == seq_len),
                      true,
                      phi::errors::InvalidArgument(
                          "The seq_len and head_dim of sin and cos "
                          "must be the same as those of q. But recieved sin's "
                          "shape is {%s}, q's shape is {%s}.",
                          sin_dims,
                          q.dims()));

    sin_cos_data[0] = sin->data<T>();
    sin_cos_data[1] = cos->data<T>();

    flag_sin_cos = true;
  }

  bool flag_position_ids = false;
  if (position_ids.get_ptr()) {
    position_ids_data = position_ids->data<int64_t>();

    flag_position_ids = true;
  }

  int sign = 1;
  if (use_neox_rotary_style) {
    VectorizedFusedRopeWithRotateEveryTwoKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     position_ids_data,
                                     flag_sin_cos,
                                     flag_position_ids,
                                     sign,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs,
                                     div_c);
  } else {
    VectorizedFusedRopeWithRotateHalfKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     position_ids_data,
                                     flag_sin_cos,
                                     flag_position_ids,
                                     sign,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs,
                                     div_c);
  }
}
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_rotary_position_embedding,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRopeKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
