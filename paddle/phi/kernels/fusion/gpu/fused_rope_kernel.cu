#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
namespace phi {
namespace fusion {

template <typename T, typename MPType, int VecSize = 2>
__global__ void VectorizedFusedRopeWithSinCosKernel(
    phi::Array<const T*, 3> ins_data,
    phi::Array<const T*, 2> sin_cos_data,
    int batch_size,
    int seq_len,
    int num_heads,
    int head_dim,
    phi::Array<T*, 3> outs_data,
    int num_inputs) {
  int index = (blockIdx.x * blockDim.x + threadIdx.x) * VecSize;
  int stride = gridDim.x * blockDim.x * VecSize;
  int size = batch_size * seq_len * num_heads * head_dim;
  MPType sin_value[VecSize];
  MPType cos_value[VecSize];
  MPType result[VecSize];
  T store[VecSize];
  using VecType = phi::AlignedVector<T, VecSize>;
  constexpr int kVectorsPerThread = VecSize / 2;

  for (; index < size; index += stride) {
#pragma unroll
    for (int nx = 0; nx < VecSize; ++nx) {
      int index_wc = (index + nx) % (seq_len * num_heads * head_dim);
      int pos_seq = index_wc / (num_heads * head_dim);
      int pos_head = index_wc % head_dim;
      int index_sc = pos_seq * head_dim + pos_head;
      const T* sin_input = sin_cos_data[0] + index_sc;
      const T* cos_input = sin_cos_data[1] + index_sc;

      sin_value[nx] = static_cast<MPType>(sin_input[0]);
      cos_value[nx] = static_cast<MPType>(cos_input[0]);
    }

#pragma unroll
    for (int iter = 0; iter < 3; iter++) {
      if (iter > num_inputs) break;
      const T* input = ins_data[iter] + index;
      VecType* out = reinterpret_cast<VecType*>(outs_data[iter] + index);

#pragma unroll
      for (int nx = 0; nx < kVectorsPerThread; ++nx) {
        int pr_index = nx * 2;
        int ls_index = pr_index + 1;

        MPType p0 = static_cast<MPType>(input[pr_index]);
        MPType p1 = static_cast<MPType>(input[ls_index]);

        result[pr_index] = cos_value[pr_index] * p0;
        result[pr_index] -= sin_value[pr_index] * p1;

        result[ls_index] = sin_value[ls_index] * p0;
        result[ls_index] += cos_value[ls_index] * p1;

        store[pr_index] = static_cast<T>(result[pr_index]);
        store[ls_index] = static_cast<T>(result[ls_index]);
      }
      out[0] = *(reinterpret_cast<VecType*>(store));
    }
  }
}

template <typename T, typename MPType, int VecSize = 2>
__global__ void VectorizedFusedRopeKernel(phi::Array<const T*, 3> ins_data,
                                          int batch_size,
                                          int seq_len,
                                          int num_heads,
                                          int head_dim,
                                          phi::Array<T*, 3> outs_data,
                                          int num_inputs,
                                          MPType div_c) {
  int index = (blockIdx.x * blockDim.x + threadIdx.x) * VecSize;
  int stride = gridDim.x * blockDim.x * VecSize;
  int size = batch_size * seq_len * num_heads * head_dim;
  MPType sin_value[VecSize];
  MPType cos_value[VecSize];
  MPType result[VecSize];
  T store[VecSize];
  using VecType = phi::AlignedVector<T, VecSize>;
  constexpr int kVectorsPerThread = VecSize / 2;

  for (; index < size; index += stride) {
#pragma unroll
    for (int nx = 0; nx < VecSize; ++nx) {
      // get sin_index and cos_index
      int index_wc = (index + nx) % (seq_len * num_heads * head_dim);
      int pos_seq = index_wc / (num_heads * head_dim);
      MPType idx = static_cast<MPType>((index_wc % head_dim) / 2 * 2.0);
      MPType indicses =
          static_cast<MPType>(1) /
          pow(static_cast<MPType>(10000), idx * static_cast<MPType>(div_c));
      MPType value = pos_seq * indicses;
      sin_value[nx] = sin(value);
      cos_value[nx] = cos(value);
    }

#pragma unroll
    for (int iter = 0; iter < 3; iter++) {
      if (iter > num_inputs) break;
      const T* input = ins_data[iter] + index;
      VecType* out = reinterpret_cast<VecType*>(outs_data[iter] + index);

#pragma unroll
      for (int nx = 0; nx < kVectorsPerThread; ++nx) {
        int pr_index = nx * 2;
        int ls_index = pr_index + 1;

        MPType p0 = static_cast<MPType>(input[pr_index]);
        MPType p1 = static_cast<MPType>(input[ls_index]);

        result[pr_index] = cos_value[pr_index] * p0;
        result[pr_index] -= sin_value[pr_index] * p1;

        result[ls_index] = sin_value[ls_index] * p0;
        result[ls_index] += cos_value[ls_index] * p1;

        store[pr_index] = static_cast<T>(result[pr_index]);
        store[ls_index] = static_cast<T>(result[ls_index]);
      }
      out[0] = *(reinterpret_cast<VecType*>(store));
    }
  }
}

template <typename T, typename Context>
void FusedRopeKernel(const Context& dev_ctx,
                     const DenseTensor& q,
                     const paddle::optional<DenseTensor>& k,
                     const paddle::optional<DenseTensor>& v,
                     const paddle::optional<DenseTensor>& sin,
                     const paddle::optional<DenseTensor>& cos,
                     DenseTensor* out_q,
                     DenseTensor* out_k,
                     DenseTensor* out_v) {
  int numel = q.numel();
  if (numel <= 0) return;
  dev_ctx.template Alloc<T>(out_q);
  out_q->Resize(q.dims());
  // small size for broadcast
  auto batch_size = q.dims()[0];
  auto num_heads = q.dims()[2];
  auto head_dim = q.dims()[3];
  auto seq_len = q.dims()[1];
  PADDLE_ENFORCE_NE(head_dim % 2,
                    1,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 2."));

  constexpr const int vec_size = 2;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

  int grid = config.block_per_grid.x;
  int block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;

  ins_data[0] = q.data<T>();
  outs_data[0] = out_q->data<T>();
  int num_inputs = 0;

  if (k.get_ptr()) {
    dev_ctx.template Alloc<T>(out_k);
    out_k->Resize(q.dims());
    ins_data[1] = k->data<T>();
    outs_data[1] = out_k->data<T>();
    num_inputs++;
  }

  if (v.get_ptr()) {
    dev_ctx.template Alloc<T>(out_v);
    out_v->Resize(q.dims());
    ins_data[2] = v->data<T>();
    outs_data[2] = out_v->data<T>();
    num_inputs++;
  }

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType div_c = static_cast<MPType>(1.0f / head_dim);

  if (sin.get_ptr() && cos.get_ptr()) {
    PADDLE_ENFORCE_EQ(sin.get_ptr()->dims(),
                      cos.get_ptr()->dims(),
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be the same."));
    auto sin_dims = sin.get_ptr()->dims();
    int dims_size = sin_dims.size();
    PADDLE_ENFORCE_NE((dims_size == 2 || dims_size == 4),
                      false,
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be 2 or 4."));
    if (dims_size == 4) {
      PADDLE_ENFORCE_NE(
          (sin_dims[0] == 1 && sin_dims[1] == 1),
          false,
          phi::errors::InvalidArgument(
              "The batch_size and num_heads of sin and cos must be 1."));
    }
    PADDLE_ENFORCE_NE(
        (sin_dims[dims_size - 1] == head_dim &&
         sin_dims[dims_size - 2] == seq_len),
        false,
        phi::errors::InvalidArgument("The seq_len and head_dim of sin and cos "
                                     "must be the same as those of q."));

    sin_cos_data[0] = sin->data<T>();
    sin_cos_data[1] = cos->data<T>();

    VectorizedFusedRopeWithSinCosKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs);
  } else {
    VectorizedFusedRopeKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs,
                                     div_c);
  }
}
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_rotary_position_embedding,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRopeKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
