#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/fusion/gpu/fused_rope_utils.h"

namespace phi {
namespace fusion {

template <typename T, typename Context>
void FusedRopeKernel(const Context& dev_ctx,
                     const DenseTensor& q,
                     const paddle::optional<DenseTensor>& k,
                     const paddle::optional<DenseTensor>& v,
                     const paddle::optional<DenseTensor>& sin,
                     const paddle::optional<DenseTensor>& cos,
                     const paddle::optional<DenseTensor>& position_ids,
                     bool use_neox_rotary_style,
                     bool time_major,
                     DenseTensor* out_q,
                     DenseTensor* out_k,
                     DenseTensor* out_v) {
  int64_t numel = q.numel();
  if (numel <= 0) return;
  dev_ctx.template Alloc<T>(out_q);

  phi::Array<int64_t, 3> inputs_num_heads;

  // q.shape: [seq_len, batch_size, num_heads, head_dim] if time_major else
  // [batch_size, seq_len, num_heads, head_dim]
  auto batch_size = time_major ? q.dims()[1] : q.dims()[0];
  auto seq_len = time_major ? q.dims()[0] : q.dims()[1];
  inputs_num_heads[0] = q.dims()[2];
  auto head_dim = q.dims()[3];

  PADDLE_ENFORCE_EQ(head_dim % 2,
                    0,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 2."));

  constexpr const int vec_size = 2;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

  int64_t grid = config.block_per_grid.x;
  int64_t block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;
  const int64_t* position_ids_data = NULL;

  ins_data[0] = q.data<T>();
  outs_data[0] = out_q->data<T>();
  int num_inputs = 1;

  if (k) {
    dev_ctx.template Alloc<T>(out_k);
    ins_data[num_inputs] = k->data<T>();
    outs_data[num_inputs] = out_k->data<T>();
    inputs_num_heads[num_inputs] = k->dims()[2];
    num_inputs++;
  }

  if (v) {
    dev_ctx.template Alloc<T>(out_v);
    ins_data[num_inputs] = v->data<T>();
    outs_data[num_inputs] = out_v->data<T>();
    inputs_num_heads[num_inputs] = v->dims()[2];
    num_inputs++;
  }

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType div_c = static_cast<MPType>(1.0f / head_dim);

  bool flag_sin_cos = false;
  auto sin_dims = sin.get_ptr()->dims();
  if (sin.get_ptr() && cos.get_ptr()) {
    PADDLE_ENFORCE_EQ(sin.get_ptr()->dims(),
                      cos.get_ptr()->dims(),
                      phi::errors::InvalidArgument(
                          "The dims of sin and cos must be the same. But "
                          "received sin's dims is {%s}, cos's dims is {%s}.",
                          sin.get_ptr()->dims(),
                          cos.get_ptr()->dims()));

    int dims_size = sin_dims.size();
    PADDLE_ENFORCE_EQ(
        (dims_size == 2 || dims_size == 4),
        true,
        phi::errors::InvalidArgument("The dims of sin and cos is expected to "
                                     "be 2 or 4, but received %d.",
                                     dims_size));
    if (dims_size == 4) {
      // sin.shape: [batch_size, seq_len, 1, head_dim]
      PADDLE_ENFORCE_EQ(
          (sin_dims[0] == 1 || sin_dims[0] == batch_size),
          true,
          phi::errors::InvalidArgument("The batch_size of sin and cos must be "
                                       "1 or equal to batch_size."));
      PADDLE_ENFORCE_EQ((sin_dims[2] == 1),
                        true,
                        phi::errors::InvalidArgument(
                            "The num_heads of sin and cos must be 1."));
    }
    int sin_seq_len_dim = (dims_size) == 4 ? 1 : 0;

    if (position_ids) {
      PADDLE_ENFORCE_EQ(
          (sin_dims[dims_size - 1] == head_dim &&
           sin_dims[sin_seq_len_dim] >= seq_len),
          true,
          phi::errors::InvalidArgument(
              "The seq_len of sin and cos must be greater than or equal to "
              "this of q. The head_dim of sin and cos must be the same as this "
              "of q. But received sin's "
              "shape is {%s}, q's shape is {%s}.",
              sin_dims,
              q.dims()));

      auto position_ids_dims = position_ids.get_ptr()->dims();
      PADDLE_ENFORCE_EQ(position_ids_dims.size(),
                        2,
                        phi::errors::InvalidArgument(
                            "The dims of position_ids is expected to "
                            "be 2, but received %d.",
                            position_ids_dims.size()));

      PADDLE_ENFORCE_EQ(
          (position_ids_dims[0] == batch_size &&
           position_ids_dims[1] == seq_len),
          true,
          phi::errors::InvalidArgument(
              "The batch_size and seq_len of position_ids must be the same as "
              "those of q. But received position_ids's "
              "shape is {%s}, q's shape is {%s}.",
              position_ids_dims,
              q.dims()));

      position_ids_data = position_ids->data<int64_t>();
    } else {
      PADDLE_ENFORCE_EQ(
          (sin_dims[dims_size - 1] == head_dim &&
           sin_dims[sin_seq_len_dim] == seq_len),
          true,
          phi::errors::InvalidArgument(
              "The seq_len and head_dim of sin and cos "
              "must be the same as those of q. But received sin's "
              "shape is {%s}, q's shape is {%s}.",
              sin_dims,
              q.dims()));
    }

    sin_cos_data[0] = sin->data<T>();
    sin_cos_data[1] = cos->data<T>();

    flag_sin_cos = true;
  }

  bool is_same_num_heads = true;
  auto prev_num_heads = inputs_num_heads[0];
  for (int i = 1; i < num_inputs; ++i) {
    if (prev_num_heads != inputs_num_heads[i]) {
      is_same_num_heads = false;
      break;
    }
    prev_num_heads = inputs_num_heads[i];
  }

  int sign = 1;
  VectorizedFusedRopeCudaKernelFunc<T, MPType, vec_size> kernel_func =
      use_neox_rotary_style
          ? VectorizedFusedRopeWithRotateEveryTwoKernel<T, MPType, vec_size>
          : VectorizedFusedRopeWithRotateHalfKernel<T, MPType, vec_size>;

  if (is_same_num_heads) {
    int64_t batch_stride = time_major ? q.strides()[1] : q.strides()[0];
    int64_t seq_stride = time_major ? q.strides()[0] : q.strides()[1];
    kernel_func<<<grid, block, 0, stream>>>(ins_data,
                                            sin_cos_data,
                                            position_ids_data,
                                            flag_sin_cos,
                                            sign,
                                            sin_dims[0],
                                            batch_size,
                                            seq_len,
                                            inputs_num_heads[0],
                                            head_dim,
                                            batch_stride,
                                            seq_stride,
                                            outs_data,
                                            num_inputs,
                                            div_c);
  } else {
    // Multi Query Attention (MQA) or Group Query Attention (GQA)
    PADDLE_ENFORCE_EQ(
        (inputs_num_heads[0] != inputs_num_heads[num_inputs - 1]) &&
            (inputs_num_heads[0] % inputs_num_heads[num_inputs - 1] == 0),
        true,
        phi::errors::InvalidArgument(
            "The MQA or GQA mode is entered, when the number of heads of qkv "
            "is not exactly the same two by two. This mode requires "
            "num_heads of q to be divisible by k,v."
            "But recieved num_heads of q is %d, num_heads of k,v is %d",
            inputs_num_heads[0],
            inputs_num_heads[num_inputs - 1]));

    if (k.get_ptr() && v.get_ptr()) {
      PADDLE_ENFORCE_EQ(
          inputs_num_heads[1] == inputs_num_heads[2],
          true,
          phi::errors::InvalidArgument(
              "The num_heads of k must be equal to the num_heads of v when v "
              "is not none."
              "But recieved num_heads of k is %d, num_heads of v is %d",
              inputs_num_heads[1],
              inputs_num_heads[2]));
    }
    // rotary position embedding Q
    int64_t batch_stride_q = time_major ? q.strides()[1] : q.strides()[0];
    int64_t seq_stride_q = time_major ? q.strides()[0] : q.strides()[1];

    kernel_func<<<grid, block, 0, stream>>>(ins_data,
                                            sin_cos_data,
                                            position_ids_data,
                                            flag_sin_cos,
                                            sign,
                                            sin_dims[0],
                                            batch_size,
                                            seq_len,
                                            inputs_num_heads[0],
                                            head_dim,
                                            batch_stride_q,
                                            seq_stride_q,
                                            outs_data,
                                            1,
                                            div_c);

    // rotary position embedding K,V
    phi::Array<const T*, 3> input_kv{ins_data[1], ins_data[2], nullptr};
    phi::Array<T*, 3> out_kv{outs_data[1], outs_data[2], nullptr};
    int64_t batch_stride_kv = time_major
                                  ? inputs_num_heads[1] * head_dim
                                  : seq_len * inputs_num_heads[1] * head_dim;
    int64_t seq_stride_kv = time_major
                                ? batch_size * inputs_num_heads[1] * head_dim
                                : inputs_num_heads[1] * head_dim;

    kernel_func<<<grid, block, 0, stream>>>(input_kv,
                                            sin_cos_data,
                                            position_ids_data,
                                            flag_sin_cos,
                                            sign,
                                            sin_dims[0],
                                            batch_size,
                                            seq_len,
                                            inputs_num_heads[1],
                                            head_dim,
                                            batch_stride_kv,
                                            seq_stride_kv,
                                            out_kv,
                                            num_inputs - 1,
                                            div_c);
  }
}

template <typename T, typename Context>
void FusedRope3DKernel(const Context& dev_ctx,
                       const DenseTensor& q,
                       const paddle::optional<DenseTensor>& k,
                       const paddle::optional<DenseTensor>& v,
                       const paddle::optional<DenseTensor>& sin,
                       const paddle::optional<DenseTensor>& cos,
                       DenseTensor* out_q,
                       DenseTensor* out_k,
                       DenseTensor* out_v) {
  dev_ctx.template Alloc<T>(out_q);

  phi::Array<int64_t, 3> inputs_num_heads;

  // q.shape: [batch_size, seq_len, num_heads, head_dim]
  auto batch_size = q.dims()[0];
  auto seq_len = q.dims()[1];
  inputs_num_heads[0] = q.dims()[2];
  auto head_dim = q.dims()[3];

  PADDLE_ENFORCE_EQ(head_dim % 6,
                    0,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 6."));

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  constexpr const int vec_size = 2;
  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, q.numel(), vec_size);

  int64_t grid = config.block_per_grid.x;
  int64_t block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;

  ins_data[0] = q.data<T>();
  outs_data[0] = out_q->data<T>();
  int num_inputs = 1;

  if (k) {
    dev_ctx.template Alloc<T>(out_k);
    ins_data[num_inputs] = k->data<T>();
    outs_data[num_inputs] = out_k->data<T>();
    inputs_num_heads[num_inputs] = k->dims()[2];
    num_inputs++;
  }

  if (v) {
    dev_ctx.template Alloc<T>(out_v);
    ins_data[num_inputs] = v->data<T>();
    outs_data[num_inputs] = out_v->data<T>();
    inputs_num_heads[num_inputs] = v->dims()[2];
    num_inputs++;
  }

  PADDLE_ENFORCE_EQ(
      sin.get_ptr() && cos.get_ptr(),
      true,
      phi::errors::InvalidArgument("The sin and cos should not be None."));

  PADDLE_ENFORCE_EQ(sin.get_ptr()->dims(),
                    cos.get_ptr()->dims(),
                    phi::errors::InvalidArgument(
                        "The dims of sin and cos must be the same. But "
                        "received sin's dims is {%s}, cos's dims is {%s}.",
                        sin.get_ptr()->dims(),
                        cos.get_ptr()->dims()));
  auto sin_dims = sin.get_ptr()->dims();
  int dims_size = sin_dims.size();
  PADDLE_ENFORCE_EQ(
      (dims_size == 4 || dims_size == 6),
      true,
      phi::errors::InvalidArgument("The dims of sin and cos is expected to "
                                   "be 4 or 6, but received %d.",
                                   dims_size));
  // sin.shape: [1 or batch_size, frame * height * width, 1, head_dim] or
  // sin.shape: [1 or batch_size, frame, height, width, 1, head_dim]
  PADDLE_ENFORCE_EQ(
      (sin_dims[0] == 1 || sin_dims[0] == batch_size),
      true,
      phi::errors::InvalidArgument(
          "The batch_size and num_heads of sin and cos must be 1."));
  PADDLE_ENFORCE_EQ(
      (sin_dims[dims_size - 2] == 1),
      true,
      phi::errors::InvalidArgument(
          "The batch_size and num_heads of sin and cos must be 1."));
  if (dims_size == 4) {
    PADDLE_ENFORCE_EQ((sin_dims[1] == seq_len),
                      true,
                      phi::errors::InvalidArgument(
                          "The sin.shape[1] must be equal to seq_len."));
  }
  if (dims_size == 6) {
    PADDLE_ENFORCE_EQ((sin_dims[1] * sin_dims[2] * sin_dims[3] == seq_len),
                      true,
                      phi::errors::InvalidArgument(
                          "The sin.shape[1] * sin.shape[2] * sin.shape[3] "
                          "must be equal to seq_len."));
  }
  PADDLE_ENFORCE_EQ((sin_dims[dims_size - 1] == head_dim),
                    true,
                    phi::errors::InvalidArgument(
                        "The head_dim of sin and cos "
                        "must be the same as that of q. But received sin's "
                        "shape is {%s}, q's shape is {%s}.",
                        sin_dims,
                        q.dims()));
  sin_cos_data[0] = sin->data<T>();
  sin_cos_data[1] = cos->data<T>();

  bool is_same_num_heads = true;
  auto prev_num_heads = inputs_num_heads[0];
  for (int i = 1; i < num_inputs; ++i) {
    if (prev_num_heads != inputs_num_heads[i]) {
      is_same_num_heads = false;
      break;
    }
    prev_num_heads = inputs_num_heads[i];
  }

  int sign = 1;
  if (is_same_num_heads) {
    int64_t seq_stride = q.strides()[1];
    VectorizedFusedRope3DKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     sin_dims[0],
                                     sign,
                                     batch_size,
                                     seq_len,
                                     inputs_num_heads[0],
                                     head_dim,
                                     seq_stride,
                                     outs_data,
                                     num_inputs);
  } else {
    // Multi Query Attention (MQA) or Group Query Attention (GQA)
    PADDLE_ENFORCE_EQ(
        (inputs_num_heads[0] != inputs_num_heads[num_inputs - 1]) &&
            (inputs_num_heads[0] % inputs_num_heads[num_inputs - 1] == 0),
        true,
        phi::errors::InvalidArgument(
            "The MQA or GQA mode is entered, when the number of heads of qkv "
            "is not exactly the same two by two. This mode requires "
            "num_heads of q to be divisible by k,v."
            "But recieved num_heads of q is %d, num_heads of k,v is %d",
            inputs_num_heads[0],
            inputs_num_heads[num_inputs - 1]));

    if (k.get_ptr() && v.get_ptr()) {
      PADDLE_ENFORCE_EQ(
          inputs_num_heads[1] == inputs_num_heads[2],
          true,
          phi::errors::InvalidArgument(
              "The num_heads of k must be equal to the num_heads of v when v "
              "is not none."
              "But recieved num_heads of k is %d, num_heads of v is %d",
              inputs_num_heads[1],
              inputs_num_heads[2]));
    }
    // rotary position embedding Q
    int64_t seq_stride_q = q.strides()[1];
    VectorizedFusedRope3DKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     sin_dims[0],
                                     sign,
                                     batch_size,
                                     seq_len,
                                     inputs_num_heads[0],
                                     head_dim,
                                     seq_stride_q,
                                     outs_data,
                                     1);

    // rotary position embedding K,V
    phi::Array<const T*, 3> input_kv{ins_data[1], ins_data[2], nullptr};
    phi::Array<T*, 3> out_kv{outs_data[1], outs_data[2], nullptr};
    int64_t seq_stride_kv = inputs_num_heads[1] * head_dim;

    VectorizedFusedRope3DKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(input_kv,
                                     sin_cos_data,
                                     sin_dims[0],
                                     sign,
                                     batch_size,
                                     seq_len,
                                     inputs_num_heads[1],
                                     head_dim,
                                     seq_stride_kv,
                                     out_kv,
                                     num_inputs - 1);
  }
}

}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_rotary_position_embedding,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRopeKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
PD_REGISTER_KERNEL(fused_rotary_position_embedding_3d,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRope3DKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
