#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_launch_config.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
namespace phi {
namespace fusion {
template <typename T, typename MPType, int VecSize = 2>
__global__ void VectorizedFusedRopeWithSinCosGradKernel(
    phi::Array<const T*, 3> ins_data,
    phi::Array<const T*, 2> sin_cos_data,
    int batch_size,
    int seq_len,
    int num_heads,
    int head_dim,
    phi::Array<T*, 3> outs_data,
    int num_inputs) {
  int index = (blockIdx.x * blockDim.x + threadIdx.x) * VecSize;
  int stride = gridDim.x * blockDim.x * VecSize;
  int size = batch_size * seq_len * num_heads * head_dim;
  MPType sin_value[VecSize];
  MPType cos_value[VecSize];
  MPType result[VecSize];
  T store[VecSize];
  using VecType = phi::AlignedVector<T, VecSize>;
  constexpr int kVectorsPerThread = VecSize / 2;

  for (; index < size; index += stride) {
#pragma unroll
    for (int nx = 0; nx < VecSize; ++nx) {
      int index_wc = (index + nx) % (seq_len * head_dim);
      const T* sin_input = sin_cos_data[0] + index_wc;
      const T* cos_input = sin_cos_data[1] + index_wc;

      sin_value[nx] = static_cast<MPType>(sin_input[0]);
      cos_value[nx] = static_cast<MPType>(cos_input[0]);
    }

#pragma unroll
    for (int iter = 0; iter < 3; iter++) {
      if (iter > num_inputs) break;
      const T* input = ins_data[iter] + index;
      VecType* out = reinterpret_cast<VecType*>(outs_data[iter] + index);

#pragma unroll
      for (int nx = 0; nx < kVectorsPerThread; ++nx) {
        int pr_index = nx * 2;
        int ls_index = pr_index + 1;

        MPType p0 = static_cast<MPType>(input[pr_index]);
        MPType p1 = static_cast<MPType>(input[ls_index]);
        result[pr_index] = cos_value[pr_index] * p0 + sin_value[ls_index] * p1;
        result[ls_index] = cos_value[ls_index] * p1 - sin_value[pr_index] * p0;

        store[pr_index] = static_cast<T>(result[pr_index]);
        store[ls_index] = static_cast<T>(result[ls_index]);
      }
      out[0] = *(reinterpret_cast<VecType*>(store));
    }
  }
}

template <typename T, typename MPType, int VecSize = 2>
__global__ void VectorizedFusedRopeGradKernel(phi::Array<const T*, 3> ins_data,
                                              int batch_size,
                                              int seq_len,
                                              int num_heads,
                                              int head_dim,
                                              phi::Array<T*, 3> outs_data,
                                              int num_inputs,
                                              MPType div_c) {
  int index = (blockIdx.x * blockDim.x + threadIdx.x) * VecSize;
  int stride = gridDim.x * blockDim.x * VecSize;
  int size = batch_size * seq_len * num_heads * head_dim;
  MPType sin_value[VecSize];
  MPType cos_value[VecSize];
  MPType result[VecSize];
  T store[VecSize];
  using VecType = phi::AlignedVector<T, VecSize>;
  constexpr int kVectorsPerThread = VecSize / 2;

  for (; index < size; index += stride) {
#pragma unroll
    for (int nx = 0; nx < VecSize; ++nx) {
      // get sin_index and cos_index
      int index_wc = (index + nx) % (seq_len * num_heads * head_dim);
      int pos_seq = index_wc / (num_heads * head_dim);
      MPType idx = static_cast<MPType>((index_wc % head_dim) / 2 * 2.0);
      MPType indicses =
          static_cast<MPType>(1) /
          pow(static_cast<MPType>(10000), idx * static_cast<MPType>(div_c));
      MPType value = pos_seq * indicses;
      sin_value[nx] = sin(value);
      cos_value[nx] = cos(value);
    }

#pragma unroll
    for (int iter = 0; iter < 3; iter++) {
      if (iter > num_inputs) break;
      const T* input = ins_data[iter] + index;
      VecType* out = reinterpret_cast<VecType*>(outs_data[iter] + index);

#pragma unroll
      for (int nx = 0; nx < kVectorsPerThread; ++nx) {
        int pr_index = nx * 2;
        int ls_index = pr_index + 1;

        MPType p0 = static_cast<MPType>(input[pr_index]);
        MPType p1 = static_cast<MPType>(input[ls_index]);
        result[pr_index] = cos_value[pr_index] * p0 + sin_value[ls_index] * p1;
        result[ls_index] = cos_value[ls_index] * p1 - sin_value[pr_index] * p0;

        store[pr_index] = static_cast<T>(result[pr_index]);
        store[ls_index] = static_cast<T>(result[ls_index]);
      }
      out[0] = *(reinterpret_cast<VecType*>(store));
    }
  }
}

template <typename T, typename Context>
void FusedRopeGradKernel(const Context& dev_ctx,
                         const paddle::optional<DenseTensor>& sin,
                         const paddle::optional<DenseTensor>& cos,
                         const DenseTensor& dout_q,
                         const paddle::optional<DenseTensor>& dout_k,
                         const paddle::optional<DenseTensor>& dout_v,
                         DenseTensor* dq,
                         DenseTensor* dk,
                         DenseTensor* dv) {
  int numel = dout_q.numel();
  if (numel <= 0) return;
  dev_ctx.template Alloc<T>(dq);
  dq->Resize(dout_q.dims());
  // small size for broadcast
  auto batch_size = dout_q.dims()[0];
  auto num_heads = dout_q.dims()[2];
  auto head_dim = dout_q.dims()[3];
  auto seq_len = dout_q.dims()[1];
  PADDLE_ENFORCE_NE(head_dim % 2,
                    1,
                    phi::errors::InvalidArgument(
                        "The head_dim of input must be a multiple of 2."));

  constexpr const int vec_size = 2;

  auto config =
      phi::backends::gpu::GetGpuLaunchConfig1D(dev_ctx, numel, vec_size);

  int grid = config.block_per_grid.x;
  int block = config.thread_per_block.x;
  auto stream = dev_ctx.stream();

  phi::Array<T*, 3> outs_data;
  phi::Array<const T*, 3> ins_data;
  phi::Array<const T*, 2> sin_cos_data;

  ins_data[0] = dout_q.data<T>();
  outs_data[0] = dq->data<T>();
  int num_inputs = 0;

  if (dout_k.get_ptr()) {
    dev_ctx.template Alloc<T>(dk);
    dk->Resize(dout_q.dims());
    outs_data[1] = dk->data<T>();
    ins_data[1] = dout_k->data<T>();
    num_inputs++;
  }

  if (dout_v.get_ptr()) {
    dev_ctx.template Alloc<T>(dv);
    dv->Resize(dout_q.dims());
    outs_data[2] = dv->data<T>();
    ins_data[2] = dout_v->data<T>();
    num_inputs++;
  }

  using MPType = typename phi::dtype::MPTypeTrait<T>::Type;
  MPType div_c = static_cast<MPType>(1.0f / head_dim);

  if (sin.get_ptr() && cos.get_ptr()) {
    sin_cos_data[0] = sin->data<T>();
    sin_cos_data[1] = cos->data<T>();

    VectorizedFusedRopeWithSinCosGradKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     sin_cos_data,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs);
  } else {
    VectorizedFusedRopeGradKernel<T, MPType, vec_size>
        <<<grid, block, 0, stream>>>(ins_data,
                                     batch_size,
                                     seq_len,
                                     num_heads,
                                     head_dim,
                                     outs_data,
                                     num_inputs,
                                     div_c);
  }
}

}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_rotary_position_embedding_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedRopeGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16){};
