#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/common/flags.h"
#include "paddle/phi/backends/gpu/cuda/cudnn_helper.h"
#include "paddle/phi/backends/gpu/gpu_dnn.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/common/int_array.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/expand_kernel.h"
#include "paddle/phi/kernels/gpudnn/mha_cudnn_frontend.h"

namespace phi {
namespace fusion {

__global__ void set_rng_state(std::pair<uint64_t, uint64_t> seed_offset,
                              int64_t* rng_state_ptr) {
  rng_state_ptr[0] = static_cast<int64_t>(seed_offset.first);
  rng_state_ptr[1] = static_cast<int64_t>(seed_offset.second);
}

const std::map<std::string, MHA_Bias_Type> kBiasTypeMap = {
    {"none", MHA_Bias_Type::NO_BIAS},
    {"pre_scale_bias", MHA_Bias_Type::PRE_SCALE_BIAS},
    {"post_scale_bias", MHA_Bias_Type::POST_SCALE_BIAS}};

const std::map<std::string, MHA_Mask_Type> kMaskTypeMap = {
    {"causal", MHA_Mask_Type::CAUSAL_MASK},
    {"padding", MHA_Mask_Type::PADDING_MASK},
    {"none", MHA_Mask_Type::NO_MASK},
    {"padding_causal", MHA_Mask_Type::PADDING_CAUSAL_MASK}};

// get cuDNN data type
cudnn_frontend::DataType_t get_cudnn_fe_dtype(const phi::DataType& t) {
  switch (t) {
    case phi::DataType::INT32:
      return cudnn_frontend::DataType_t::INT32;
    case phi::DataType::INT64:
      return cudnn_frontend::DataType_t::INT64;
    case phi::DataType::FLOAT16:
      return cudnn_frontend::DataType_t::HALF;
    case phi::DataType::FLOAT32:
      return cudnn_frontend::DataType_t::FLOAT;
    case phi::DataType::BFLOAT16:
      return cudnn_frontend::DataType_t::BFLOAT16;
    default:
      PADDLE_THROW(
          phi::errors::InvalidArgument("Invalid data type for cuDNN frontend"));
  }
}

template <typename T, typename Context>
void FusedDotProductAttentionKernel(
    const Context& dev_ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const paddle::optional<DenseTensor>& bias,
    const paddle::optional<DenseTensor>& cu_seqlen_q,
    const paddle::optional<DenseTensor>& cu_seqlen_kv,
    float scaling_factor,
    float dropout_probability,
    bool is_training,
    const std::string& mask_type_str,
    const std::string& bias_type_str,
    DenseTensor* out,
    DenseTensor* softmax_out,
    DenseTensor* rng_state) {
  PADDLE_ENFORCE_GE(dev_ctx.GetComputeCapability(),
                    80,
                    phi::errors::PreconditionNotMet(
                        "This op only supports Ampere and later devices, "
                        "but got compute capability: %d.",
                        dev_ctx.GetComputeCapability()));
  auto cudnn_version = phi::backends::gpu::DnnVersion();
  PADDLE_ENFORCE_GE(cudnn_version,
                    8906,
                    phi::errors::PreconditionNotMet(
                        "This op only supports CUDNN version >= 8906, "
                        "but got %d.",
                        cudnn_version));
  PADDLE_ENFORCE_EQ(
      (cu_seqlen_q.get_ptr() != nullptr && cu_seqlen_kv.get_ptr() != nullptr) ||
          (cu_seqlen_q.get_ptr() == nullptr &&
           cu_seqlen_kv.get_ptr() == nullptr),
      true,
      phi::errors::InvalidArgument(
          "cu_seqlen_q and cu_seqlen_kv must be both set or both not set"));

  // allocate output variables
  dev_ctx.template Alloc<T>(out);
  dev_ctx.template Alloc<float>(softmax_out);
  dev_ctx.template Alloc<int64_t>(rng_state);

  // get handles
  auto handle = dev_ctx.cudnn_handle();

  auto tensor_dtype = get_cudnn_fe_dtype(q.dtype());
  bool is_type_supported =
      (tensor_dtype == cudnn_frontend::DataType_t::HALF ||
       tensor_dtype == cudnn_frontend::DataType_t::BFLOAT16);
  PADDLE_ENFORCE_EQ(
      is_type_supported,
      true,
      phi::errors::InvalidArgument(
          "cuDNN fused attention Only supports FP16/BF16 currently"));
  auto mha_layout = MHA_Layout::BSHD_BSHD_BSHD;
  auto bias_type = MHA_Bias_Type::NO_BIAS;
  auto mask_type = MHA_Mask_Type::NO_MASK;
  auto bias_type_iter = kBiasTypeMap.find(bias_type_str);
  if (bias_type_iter != kBiasTypeMap.end()) {
    bias_type = bias_type_iter->second;
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Invalid bias type: %s, only support no_bias, pre_scale_bias, "
        "post_scale_bias",
        bias_type_str));
  }
  if (bias.get_ptr() == nullptr) {
    bias_type = MHA_Bias_Type::NO_BIAS;
  }
  auto mask_type_iter = kMaskTypeMap.find(mask_type_str);
  if (mask_type_iter != kMaskTypeMap.end()) {
    mask_type = mask_type_iter->second;
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Invalid mask type: %s, only support causal, padding, none, "
        "padding_causal",
        mask_type_str));
  }

  // q dim: {b, s_q, h, d};
  // k,v dim: {b, s_kv, h_kv, d};
  auto batch_size = q.dims()[0];
  auto q_seq_len = q.dims()[1];
  auto num_heads = q.dims()[2];
  auto num_heads_kv = k.dims()[2];
  auto head_size = q.dims()[3];
  auto kv_seq_len = k.dims()[1];

  // support bias shape: [b,1,s,s],[b,h,s,s],[1,1,s,s]
  size_t bias_b = 0;
  size_t bias_h = 0;
  void* bias_dev_ptr = nullptr;
  DenseTensor mask_expand;
  if (bias_type != MHA_Bias_Type::NO_BIAS) {
    bias_b = bias.get_ptr()->dims()[0];
    bias_h = bias.get_ptr()->dims()[1];
    bias_dev_ptr =
        reinterpret_cast<void*>(const_cast<T*>(bias.get_ptr()->data<T>()));

    // if bias's shape is [b,1,1,s], we treat it as an attention mask, broadcast
    // it to [b,1,s,s]
    if (bias_h == 1 && bias.get_ptr()->dims()[2] == 1) {
      std::vector<int64_t> mask_dims_vec = {
          static_cast<int64_t>(bias_b), 1, q_seq_len, kv_seq_len};
      phi::ExpandKernel<T, Context>(
          dev_ctx, *(bias.get_ptr()), IntArray(mask_dims_vec), &mask_expand);
      bias_dev_ptr = reinterpret_cast<void*>(mask_expand.data<T>());
    }
  }

  auto gen_cuda = dev_ctx.GetGenerator();
  const int rng_elts_per_thread = 16;
  auto seed_offset = gen_cuda->IncrementOffset(rng_elts_per_thread);
  set_rng_state<<<1, 1, 0, dev_ctx.stream()>>>(
      seed_offset, static_cast<int64_t*>(rng_state->data<int64_t>()));

  void* q_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(q.data<T>()));
  void* k_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(k.data<T>()));
  void* v_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(v.data<T>()));
  void* out_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(out->data<T>()));
  void* softmax_out_dev_ptr =
      reinterpret_cast<void*>(const_cast<float*>(softmax_out->data<float>()));
  // rng_state: {seed, offset}
  void* seed_dev_ptr =
      reinterpret_cast<void*>(const_cast<int64_t*>(rng_state->data<int64_t>()));
  void* offset_dev_ptr = reinterpret_cast<void*>(
      const_cast<int64_t*>(rng_state->data<int64_t>()) + 1);
  void* cu_seqlen_q_dev_ptr = nullptr;
  void* cu_seqlen_kv_dev_ptr = nullptr;
  if (cu_seqlen_q.get_ptr() != nullptr && cu_seqlen_kv.get_ptr() != nullptr) {
    cu_seqlen_q_dev_ptr = reinterpret_cast<void*>(
        const_cast<int32_t*>(cu_seqlen_q.get_ptr()->data<int32_t>()));
    cu_seqlen_kv_dev_ptr = reinterpret_cast<void*>(
        const_cast<int32_t*>(cu_seqlen_kv.get_ptr()->data<int32_t>()));
  }
  size_t workspace_size = 0;
  // call the first time to get the workspace size
  fused_attn_arbitrary_seqlen_fwd_impl(batch_size,
                                       num_heads,
                                       num_heads_kv,
                                       q_seq_len,
                                       kv_seq_len,
                                       head_size,
                                       bias_b,
                                       bias_h,
                                       is_training,
                                       scaling_factor,
                                       dropout_probability,
                                       mha_layout,
                                       bias_type,
                                       mask_type,
                                       q_dev_ptr,
                                       k_dev_ptr,
                                       v_dev_ptr,
                                       bias_dev_ptr,
                                       softmax_out_dev_ptr,
                                       out_dev_ptr,
                                       seed_dev_ptr,
                                       offset_dev_ptr,
                                       cu_seqlen_q_dev_ptr,
                                       cu_seqlen_kv_dev_ptr,
                                       tensor_dtype,
                                       nullptr,
                                       &workspace_size,
                                       dev_ctx);
  DenseTensor workspace;
  workspace.Resize({static_cast<int64_t>(workspace_size) > 0
                        ? static_cast<int64_t>(workspace_size)
                        : 1});
  dev_ctx.template Alloc<int8_t>(&workspace);
  // call the second time to excute the kernel
  fused_attn_arbitrary_seqlen_fwd_impl(
      batch_size,
      num_heads,
      num_heads_kv,
      q_seq_len,
      kv_seq_len,
      head_size,
      bias_b,
      bias_h,
      is_training,
      scaling_factor,
      dropout_probability,
      mha_layout,
      bias_type,
      mask_type,
      q_dev_ptr,
      k_dev_ptr,
      v_dev_ptr,
      bias_dev_ptr,
      softmax_out_dev_ptr,
      out_dev_ptr,
      seed_dev_ptr,
      offset_dev_ptr,
      cu_seqlen_q_dev_ptr,
      cu_seqlen_kv_dev_ptr,
      tensor_dtype,
      reinterpret_cast<void*>(workspace.data<int8_t>()),
      &workspace_size,
      dev_ctx);
}

template <typename T, typename Context>
void FusedDotProductAttentionGradKernel(
    const Context& dev_ctx,
    const DenseTensor& q,
    const DenseTensor& k,
    const DenseTensor& v,
    const paddle::optional<DenseTensor>& bias,
    const paddle::optional<DenseTensor>& cu_seqlen_q,
    const paddle::optional<DenseTensor>& cu_seqlen_kv,
    const DenseTensor& O,
    const DenseTensor& softmax_out,
    const DenseTensor& rng_state,
    const DenseTensor& dO,
    float scaling_factor,
    float dropout_probability,
    const std::string& mask_type_str,
    const std::string& bias_type_str,
    DenseTensor* q_grad,
    DenseTensor* k_grad,
    DenseTensor* v_grad,
    DenseTensor* bias_grad) {
  auto sm_arch = dev_ctx.GetComputeCapability();
  PADDLE_ENFORCE_GE(sm_arch,
                    80,
                    phi::errors::PreconditionNotMet(
                        "This op only supports Ampere and later devices, "
                        "but got compute capability: %d.",
                        dev_ctx.GetComputeCapability()));
  auto cudnn_version = phi::backends::gpu::DnnVersion();
  PADDLE_ENFORCE_GE(cudnn_version,
                    8906,
                    phi::errors::PreconditionNotMet(
                        "This op only supports CUDNN version >= 8906, "
                        "but got %d.",
                        cudnn_version));

  // allocate output variables
  dev_ctx.template Alloc<T>(q_grad);
  dev_ctx.template Alloc<T>(k_grad);
  dev_ctx.template Alloc<T>(v_grad);

  // get handles
  auto handle = dev_ctx.cudnn_handle();

  auto tensor_dtype = get_cudnn_fe_dtype(q.dtype());
  bool is_type_supported =
      (tensor_dtype == cudnn_frontend::DataType_t::HALF ||
       tensor_dtype == cudnn_frontend::DataType_t::BFLOAT16);
  PADDLE_ENFORCE_EQ(
      is_type_supported,
      true,
      phi::errors::InvalidArgument(
          "cuDNN fused attention Only supports FP16/BF16 currently"));
  auto mha_layout = MHA_Layout::BSHD_BSHD_BSHD;
  auto bias_type = MHA_Bias_Type::NO_BIAS;
  auto mask_type = MHA_Mask_Type::NO_MASK;
  auto bias_type_iter = kBiasTypeMap.find(bias_type_str);
  if (bias_type_iter != kBiasTypeMap.end()) {
    bias_type = bias_type_iter->second;
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Invalid bias type: %s, only support no_bias, pre_scale_bias, "
        "post_scale_bias",
        bias_type_str));
  }
  if (bias.get_ptr() == nullptr) {
    bias_type = MHA_Bias_Type::NO_BIAS;
  }
  auto mask_type_iter = kMaskTypeMap.find(mask_type_str);
  if (mask_type_iter != kMaskTypeMap.end()) {
    mask_type = mask_type_iter->second;
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "Invalid mask type: %s, only support causal, padding, none, "
        "padding_causal",
        mask_type_str));
  }

  // q dim: {b, s_q, h, d};
  // k, v dim: {b, s_kv, h_kv, d};
  auto batch_size = q.dims()[0];
  auto q_seq_len = q.dims()[1];
  auto num_heads = q.dims()[2];
  auto num_heads_kv = k.dims()[2];
  auto head_size = q.dims()[3];
  auto kv_seq_len = k.dims()[1];

  // bias dim: {b, h, s_q, s_kv}
  size_t bias_b = 0;
  size_t bias_h = 0;
  void* bias_dev_ptr = nullptr;
  void* dbias_dev_ptr = nullptr;
  DenseTensor mask_expand;
  if (bias_type != MHA_Bias_Type::NO_BIAS) {
    bias_dev_ptr =
        reinterpret_cast<void*>(const_cast<T*>(bias.get_ptr()->data<T>()));

    if (bias_grad != nullptr) {
      bias_b = bias_grad->dims()[0];
      bias_h = bias_grad->dims()[1];
      dev_ctx.template Alloc<T>(bias_grad);
      dbias_dev_ptr =
          reinterpret_cast<void*>(const_cast<T*>(bias_grad->data<T>()));
    } else {
      bias_b = bias.get_ptr()->dims()[0];
      bias_h = bias.get_ptr()->dims()[1];
    }

    // if bias's shape is [b,1,1,s], we treat it as an attention mask, broadcast
    // it to [b,1,s,s]
    if (bias_h == 1 && bias.get_ptr()->dims()[2] == 1) {
      std::vector<int64_t> mask_dims_vec = {
          static_cast<int64_t>(bias_b), 1, q_seq_len, kv_seq_len};
      phi::ExpandKernel<T, Context>(
          dev_ctx, *(bias.get_ptr()), IntArray(mask_dims_vec), &mask_expand);
      bias_dev_ptr = reinterpret_cast<void*>(mask_expand.data<T>());
    }
  }

  void* q_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(q.data<T>()));
  void* k_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(k.data<T>()));
  void* v_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(v.data<T>()));
  void* dq_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(q_grad->data<T>()));
  void* dk_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(k_grad->data<T>()));
  void* dv_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(v_grad->data<T>()));
  void* o_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(O.data<T>()));
  void* do_dev_ptr = reinterpret_cast<void*>(const_cast<T*>(dO.data<T>()));
  void* softmax_out_dev_ptr =
      reinterpret_cast<void*>(const_cast<float*>(softmax_out.data<float>()));
  void* seed_dev_ptr =
      reinterpret_cast<void*>(const_cast<int64_t*>(rng_state.data<int64_t>()));
  void* offset_dev_ptr = reinterpret_cast<void*>(
      const_cast<int64_t*>(rng_state.data<int64_t>()) + 1);
  void* cu_seqlen_q_dev_ptr = nullptr;
  void* cu_seqlen_kv_dev_ptr = nullptr;
  if (cu_seqlen_q.get_ptr() != nullptr && cu_seqlen_kv.get_ptr() != nullptr) {
    cu_seqlen_q_dev_ptr = reinterpret_cast<void*>(
        const_cast<int32_t*>(cu_seqlen_q.get_ptr()->data<int32_t>()));
    cu_seqlen_kv_dev_ptr = reinterpret_cast<void*>(
        const_cast<int32_t*>(cu_seqlen_kv.get_ptr()->data<int32_t>()));
  }

  size_t workspace_size = 0;
  // call the first time to get the workspace size
  fused_attn_arbitrary_seqlen_bwd_impl(batch_size,
                                       num_heads,
                                       num_heads_kv,
                                       q_seq_len,
                                       kv_seq_len,
                                       head_size,
                                       bias_b,
                                       bias_h,
                                       scaling_factor,
                                       dropout_probability,
                                       mha_layout,
                                       bias_type,
                                       mask_type,
                                       q_dev_ptr,
                                       k_dev_ptr,
                                       v_dev_ptr,
                                       o_dev_ptr,
                                       softmax_out_dev_ptr,
                                       bias_dev_ptr,
                                       dq_dev_ptr,
                                       dk_dev_ptr,
                                       dv_dev_ptr,
                                       do_dev_ptr,
                                       dbias_dev_ptr,
                                       seed_dev_ptr,
                                       offset_dev_ptr,
                                       cu_seqlen_q_dev_ptr,
                                       cu_seqlen_kv_dev_ptr,
                                       tensor_dtype,
                                       nullptr,
                                       &workspace_size,
                                       dev_ctx);

  DenseTensor workspace;
  workspace.Resize({static_cast<int64_t>(workspace_size) > 0
                        ? static_cast<int64_t>(workspace_size)
                        : 1});
  dev_ctx.template Alloc<int8_t>(&workspace);
  fused_attn_arbitrary_seqlen_bwd_impl(
      batch_size,
      num_heads,
      num_heads_kv,
      q_seq_len,
      kv_seq_len,
      head_size,
      bias_b,
      bias_h,
      scaling_factor,
      dropout_probability,
      mha_layout,
      bias_type,
      mask_type,
      q_dev_ptr,
      k_dev_ptr,
      v_dev_ptr,
      o_dev_ptr,
      softmax_out_dev_ptr,
      bias_dev_ptr,
      dq_dev_ptr,
      dk_dev_ptr,
      dv_dev_ptr,
      do_dev_ptr,
      dbias_dev_ptr,
      seed_dev_ptr,
      offset_dev_ptr,
      cu_seqlen_q_dev_ptr,
      cu_seqlen_kv_dev_ptr,
      tensor_dtype,
      reinterpret_cast<void*>(workspace.data<int8_t>()),
      &workspace_size,
      dev_ctx);
}

}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(fused_dot_product_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedDotProductAttentionKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(fused_dot_product_attention_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::FusedDotProductAttentionGradKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
