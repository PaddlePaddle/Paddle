
#include <hip/hip_runtime.h>
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_CUDA
#include <xxhash.h>

#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <limits>
#include <memory>
#include <unordered_map>

#include "paddle/phi/backends/dynload/cudnn.h"
#include "paddle/phi/backends/gpu/cuda/cudnn_desc.h"
#include "paddle/phi/common/backend.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/kernels/impl/conv_cudnn_impl.h"
#include "paddle/utils/optional.h"

namespace phi {
namespace fusion {

namespace {
class CudnnConvDescManager {
 public:
  static CudnnConvDescManager* Instance() {
    static CudnnConvDescManager global;
    return &global;
  }

  struct CudnnCacheInfo {
    phi::backends::gpu::TensorDescriptor x_desc;
    phi::backends::gpu::FilterDescriptor w_desc;
    phi::backends::gpu::TensorDescriptor b_desc;
    phi::backends::gpu::TensorDescriptor o_desc;
    phi::backends::gpu::ConvolutionDescriptor conv_desc;
    phi::backends::gpu::ActivationDescriptor act_desc;
    size_t workspace_size;
    cudnnConvolutionFwdAlgo_t algo;

    std::vector<int> paddings;
    std::vector<int> dilations;
    std::vector<int> input_pad;
    std::vector<int> new_input_shape_vec;
    bool is_sys_pad;
  };

  CudnnCacheInfo* GetCudnnCacheInfo(
      const std::vector<int>& input_dims,
      const std::vector<int>& filter_dims,
      const std::vector<int>& bias_dims,
      const std::vector<int>& output_dims,
      const std::vector<int>& paddings,
      const std::vector<int>& strides,
      const std::vector<int>& dilations,
      phi::DataType input_dtype,
      int groups,
      cudnnDataType_t dtype,
      cudnnTensorFormat_t format,
      const std::function<void(cudnnConvolutionFwdAlgo_t*,
                               size_t*,
                               cudnnTensorDescriptor_t,
                               cudnnFilterDescriptor_t,
                               cudnnTensorDescriptor_t,
                               cudnnConvolutionDescriptor_t)>& search_func,
      const std::string& act,
      double value_max = std::numeric_limits<double>::max()) {
    // std::hash takes about 5us, xxhash can optimize to 2.5us.
    XXH64_state_t* const state = XXH64_createState();
    if (state == nullptr) {
      CHECK(false);
    }
    XXH64_hash_t const seed = 0;
    if (XXH64_reset(state, seed) == XXH_ERROR) {
      CHECK(false);
    }
    XXH64_update(state, input_dims.data(), input_dims.size() * sizeof(int));
    XXH64_update(state, filter_dims.data(), filter_dims.size() * sizeof(int));
    XXH64_update(state, bias_dims.data(), bias_dims.size() * sizeof(int));
    // XXH64_update(state, output_dims.data(), output_dims.size() *
    // sizeof(int));
    XXH64_update(state, paddings.data(), paddings.size() * sizeof(int));
    XXH64_update(state, strides.data(), strides.size() * sizeof(int));
    XXH64_update(state, dilations.data(), dilations.size() * sizeof(int));
    XXH64_update(state, &input_dtype, sizeof(int));
    XXH64_update(state, &groups, sizeof(int));
    XXH64_update(state, &dtype, sizeof(int));
    XXH64_update(state, &format, sizeof(int));
    XXH64_update(state, &act, act.size() * sizeof(char));
    // XXH64_update(state, &value_max, sizeof(double));
    XXH64_hash_t hash_key = XXH64_digest(state);
    XXH64_freeState(state);

    if (!cudnn_conv_cache_.count(hash_key)) {
      std::lock_guard<std::mutex> lock(cache_mutex_);
      if (!cudnn_conv_cache_.count(hash_key)) {
        CudnnCacheInfo cache_info;

        cache_info.x_desc = GetTensorDescInfo(input_dims, input_dtype, format);
        cache_info.w_desc = GetFilterDescInfo(filter_dims, input_dtype, format);
        cache_info.b_desc = GetTensorDescInfo(bias_dims, input_dtype, format);
        cache_info.o_desc = GetTensorDescInfo(output_dims, input_dtype, format);
        cache_info.conv_desc =
            GetConvDescInfo(paddings, strides, dilations, groups, dtype);
        cache_info.act_desc = GetActivationDescInfo(act, value_max);

        size_t workspace_size;
        cudnnConvolutionFwdAlgo_t algo;
        search_func(&algo,
                    &workspace_size,
                    cache_info.x_desc.desc(),
                    cache_info.w_desc.desc(),
                    cache_info.o_desc.desc(),
                    cache_info.conv_desc.desc());
        cache_info.workspace_size = workspace_size;
        cache_info.algo = algo;

        cudnn_conv_cache_[hash_key] = std::move(cache_info);
      }
    }

    return &cudnn_conv_cache_.at(hash_key);
  }

  struct ConvAttrCacheInfo {
    std::vector<int> paddings;
    std::vector<int> dilations;
    std::vector<int> input_pad;
    std::vector<int> new_input_shape_vec;
    bool is_sys_pad;
  };
  ConvAttrCacheInfo* GetConvAttr(const std::vector<int>& paddings_t,
                                 const std::vector<int>& dilations_t,
                                 const std::string& padding_algorithm,
                                 const std::vector<int>& input_dims,
                                 const std::vector<int>& filter_dims,
                                 const std::vector<int>& strides,
                                 cudnnTensorFormat_t format) {
    // std::hash takes about 5us, xxhash can optimize to 2.5us.
    XXH64_state_t* const state = XXH64_createState();
    if (state == nullptr) {
      CHECK(false);
    }
    XXH64_hash_t const seed = 0;
    if (XXH64_reset(state, seed) == XXH_ERROR) {
      CHECK(false);
    }
    XXH64_update(state, paddings_t.data(), paddings_t.size() * sizeof(int));
    XXH64_update(state, dilations_t.data(), dilations_t.size() * sizeof(int));
    XXH64_update(state, input_dims.data(), input_dims.size() * sizeof(int));
    XXH64_update(state, filter_dims.data(), filter_dims.size() * sizeof(int));
    XXH64_update(state, strides.data(), strides.size() * sizeof(int));
    XXH64_update(state, &format, sizeof(int));
    XXH64_update(
        state, &padding_algorithm, padding_algorithm.size() * sizeof(char));
    XXH64_hash_t hash_key = XXH64_digest(state);
    XXH64_freeState(state);

    if (!conv_attr_cache_.count(hash_key)) {
      std::lock_guard<std::mutex> lock(attr_mutex_);
      if (!conv_attr_cache_.count(hash_key)) {
        auto printVec = [](const std::vector<int>& vec) {
          std::ostringstream os;
          for (size_t i = 0; i < vec.size(); ++i) os << vec[i] << ", ";
          return os.str();
        };
        ConvAttrCacheInfo cache;
        auto paddings = paddings_t;
        auto dilations = dilations_t;
        std::vector<int> in_data_dims(input_dims.size() - 2);
        std::vector<int> ksize(filter_dims.size() - 2);
        if (format == CUDNN_TENSOR_NHWC) {
          for (size_t i = 1; i < input_dims.size() - 1; ++i) {
            in_data_dims[i - 1] = input_dims[i];
          }
          for (size_t i = 1; i < filter_dims.size() - 1; ++i) {
            ksize[i - 1] = filter_dims[i];
          }
        } else {
          for (size_t i = 2; i < input_dims.size(); ++i) {
            in_data_dims[i - 2] = input_dims[i];
          }
          for (size_t i = 2; i < filter_dims.size(); ++i) {
            ksize[i - 2] = filter_dims[i];
          }
        }
        phi::UpdatePaddingAndDilation(&paddings,
                                      &dilations,
                                      padding_algorithm,
                                      make_ddim(in_data_dims),
                                      strides,
                                      ksize);

        int data_dim = strides.size();  // 2d or 3d
        bool is_sys_pad = funcs::IsSymmetricPadding(paddings, data_dim);
        std::vector<int> padding_common(data_dim, 0);
        if (!is_sys_pad) {
          std::vector<int> padding_diff(data_dim);
          std::vector<int> new_input_shape_vec(data_dim + 2);
          new_input_shape_vec[0] = input_dims[0];

          if (format == CUDNN_TENSOR_NCHW) {
            new_input_shape_vec[1] = input_dims[1];
          } else {
            new_input_shape_vec[data_dim + 1] = input_dims[data_dim + 1];
          }

          std::vector<int> input_pad(input_dims.size() * 2, 0);
          for (size_t i = 0; i < data_dim; ++i) {
            padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
            padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
            if (format == CUDNN_TENSOR_NCHW) {
              new_input_shape_vec[i + 2] = input_dims[i + 2] + padding_diff[i];
            } else {
              new_input_shape_vec[i + 1] = input_dims[i + 1] + padding_diff[i];
            }
            if (format == CUDNN_TENSOR_NCHW) {
              input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
              input_pad[2 * i + 4 + 1] =
                  paddings[2 * i + 1] - padding_common[i];
            } else {
              input_pad[2 * i + 2] = paddings[2 * i] - padding_common[i];
              input_pad[2 * i + 2 + 1] =
                  paddings[2 * i + 1] - padding_common[i];
            }
          }

          cache.is_sys_pad = false;
          cache.input_pad = input_pad;
          cache.new_input_shape_vec = new_input_shape_vec;
        } else {
          cache.is_sys_pad = true;
          if (paddings.size() == data_dim) {
            for (size_t i = 0; i < data_dim; ++i) {
              padding_common[i] = paddings[i];
            }
          } else {
            for (size_t i = 0; i < data_dim; ++i) {
              padding_common[i] = paddings[2 * i];
            }
          }
        }

        cache.dilations = dilations;
        cache.paddings = padding_common;
        conv_attr_cache_[hash_key] = cache;
      }
    }

    return &conv_attr_cache_.at(hash_key);
  }

 private:
  phi::backends::gpu::TensorDescriptor GetTensorDescInfo(
      const std::vector<int>& input_dims,
      phi::DataType input_dtype,
      cudnnTensorFormat_t input_format) {
    phi::backends::gpu::TensorDescriptor desc;
    desc.set(
        input_dims, input_format, backends::gpu::ToCudnnDataType(input_dtype));
    return desc;
  }

  phi::backends::gpu::FilterDescriptor GetFilterDescInfo(
      const std::vector<int>& input_dims,
      phi::DataType input_dtype,
      cudnnTensorFormat_t input_format) {
    phi::backends::gpu::FilterDescriptor desc;
    desc.set(
        input_dims, input_format, backends::gpu::ToCudnnDataType(input_dtype));
    return desc;
  }

  phi::backends::gpu::ConvolutionDescriptor GetConvDescInfo(
      const std::vector<int>& paddings,
      const std::vector<int>& strides,
      const std::vector<int>& dilations,
      int groups,
      cudnnDataType_t dtype) {
    phi::backends::gpu::ConvolutionDescriptor conv_desc;
    conv_desc.set(dtype,
                  paddings,
                  strides,
                  dilations,
                  paddle::platform::AllowTF32Cudnn(),
                  groups);
    return conv_desc;
  }

  phi::backends::gpu::ActivationDescriptor GetActivationDescInfo(
      const std::string& act,
      double value_max = std::numeric_limits<double>::max()) {
    phi::backends::gpu::ActivationDescriptor desc;
    cudnnActivationMode_t mode;
    double relu_ceiling = 0.0;
    if (act == "identity") {
      mode = CUDNN_ACTIVATION_IDENTITY;
    } else if (act == "relu") {
      mode = CUDNN_ACTIVATION_RELU;
    } else if (act == "relu6") {
      relu_ceiling = 6.0;
      mode = CUDNN_ACTIVATION_CLIPPED_RELU;
    } else if (act == "sigmoid") {
      mode = CUDNN_ACTIVATION_SIGMOID;
    } else if (act == "relux") {
      relu_ceiling = value_max;
      mode = CUDNN_ACTIVATION_CLIPPED_RELU;
    } else if (act == "tanh") {
      mode = CUDNN_ACTIVATION_TANH;
    } else {
      PADDLE_THROW(phi::errors::Unimplemented(
          "Unknown CUDNN activation string: %s.", act));
    }
    desc.set(mode, relu_ceiling);
    return desc;
  }

  std::mutex cache_mutex_;
  std::unordered_map<size_t, CudnnCacheInfo> cudnn_conv_cache_;

  std::mutex attr_mutex_;
  std::unordered_map<size_t, ConvAttrCacheInfo> conv_attr_cache_;
};
}  // namespace

template <typename T, typename Context>
void ConvFusionKernel(const Context& ctx,
                      const DenseTensor& input,
                      const DenseTensor& filter,
                      const DenseTensor& bias,
                      const paddle::optional<DenseTensor>& residual,
                      const std::vector<int>& strides,
                      const std::vector<int>& paddings_t,
                      const std::string& padding_algorithm,
                      const std::vector<int>& dilations_t,
                      int groups,
                      const std::string& data_format,
                      const std::string& activation,
                      bool exhaustive_search,
                      const std::vector<int>& channels,
                      int user_workspace_size,
                      DenseTensor* output,
                      std::vector<DenseTensor*> outs) {
  auto handle = ctx.cudnn_handle();
  ctx.template Alloc<T>(output);
  auto workspace_handle = ctx.cudnn_workspace_handle();

  exhaustive_search = FLAGS_cudnn_exhaustive_search || exhaustive_search;
  bool deterministic = FLAGS_cudnn_deterministic;
  PADDLE_ENFORCE_EQ(exhaustive_search && deterministic,
                    false,
                    phi::errors::InvalidArgument(
                        "Cann't set exhaustive_search True and "
                        "FLAGS_cudnn_deterministic True at same time."));

  size_t workspace_size_limit = 0;
  if (FLAGS_conv_workspace_size_limit > 0 || user_workspace_size > 0) {
    int64_t max_user_size =
        std::min(static_cast<int64_t>(FLAGS_conv_workspace_size_limit),
                 static_cast<int64_t>(user_workspace_size));
    workspace_size_limit = max_user_size * 1024 * 1024;
  }

  auto dtype = phi::backends::gpu::CudnnDataType<T>::type;
  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");
  // Choose NHWC or NCHW by data_format attr.
  auto compute_format = channel_last ? CUDNN_TENSOR_NHWC : CUDNN_TENSOR_NCHW;
  VLOG(3) << "Compute ConvFusionOp with cuDNN:"
          << " data_format=" << data_format << " compute_format="
          << (compute_format == CUDNN_TENSOR_NHWC ? "NHWC" : "NCHW");

  auto* conv_attr_cache = CudnnConvDescManager::Instance()->GetConvAttr(
      paddings_t,
      dilations_t,
      padding_algorithm,
      phi::vectorize<int>(input.dims()),
      phi::vectorize<int>(filter.dims()),
      strides,
      compute_format);

  DenseTensor transformed_input;
  auto unsys_pad_process = [&](const std::vector<int>& new_input_shape_vec,
                               const std::vector<int>& input_pad) {
    DDim new_input_shape(make_ddim(new_input_shape_vec));
    transformed_input.Resize(new_input_shape);
    ctx.template Alloc<T>(&transformed_input);

    const int rank = input.dims().size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(
            ctx, input_pad, input, pad_value, &transformed_input);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(
            ctx, input_pad, input, pad_value, &transformed_input);
      } break;
      default:
        PADDLE_THROW(phi::errors::InvalidArgument(
            "ConvOp only support tensors with 4 or 5 dimensions."));
    }
  };
  if (conv_attr_cache->is_sys_pad) {
    transformed_input.ShareDataWith(input);
  } else {
    unsys_pad_process(conv_attr_cache->new_input_shape_vec,
                      conv_attr_cache->input_pad);
  }

  std::vector<int> b_dims(input.dims().size(), 1);
  if (compute_format == CUDNN_TENSOR_NCHW) {
    b_dims[1] = static_cast<int>(bias.dims()[0]);
  } else {
    b_dims[input.dims().size() - 1] = static_cast<int>(bias.dims()[0]);
  }

  std::function<void(cudnnConvolutionFwdAlgo_t*,
                     size_t*,
                     cudnnTensorDescriptor_t,
                     cudnnFilterDescriptor_t,
                     cudnnTensorDescriptor_t,
                     cudnnConvolutionDescriptor_t)>
      search_func = [&](cudnnConvolutionFwdAlgo_t* cudnn_algo,
                        size_t* wks_bytes,
                        cudnnTensorDescriptor_t x_desc,
                        cudnnFilterDescriptor_t w_desc,
                        cudnnTensorDescriptor_t o_desc,
                        cudnnConvolutionDescriptor_t cudnn_conv_desc) {
        if (!exhaustive_search) {
#if CUDNN_VERSION >= 8000
          int perf_count;
          int best_algo_idx = 0;
          size_t tmp_size = 0;
          std::unique_ptr<cudnnConvolutionFwdAlgoPerf_t[]> perf_results(
              new cudnnConvolutionFwdAlgoPerf_t[phi::kNUM_CUDNN_FWD_ALGS]);
          PADDLE_ENFORCE_GPU_SUCCESS(
              phi::dynload::cudnnGetConvolutionForwardAlgorithm_v7(
                  handle,
                  x_desc,
                  w_desc,
                  cudnn_conv_desc,
                  o_desc,
                  phi::kNUM_CUDNN_FWD_ALGS,
                  &perf_count,
                  perf_results.get()));
          *cudnn_algo = (perf_results.get())[best_algo_idx].algo;
#else
          PADDLE_ENFORCE_GPU_SUCCESS(
              phi::dynload::cudnnGetConvolutionForwardAlgorithm(
                  handle,
                  x_desc,
                  w_desc,
                  cudnn_conv_desc,
                  o_desc,
                  CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
                  workspace_size_limit,
                  cudnn_algo));
#endif
          PADDLE_ENFORCE_GPU_SUCCESS(
              phi::dynload::cudnnGetConvolutionForwardWorkspaceSize(
                  handle,
                  x_desc,
                  w_desc,
                  cudnn_conv_desc,
                  o_desc,
                  *cudnn_algo,
                  wks_bytes));
        } else {
          std::array<cudnnConvolutionFwdAlgoPerf_t, phi::kNUM_CUDNN_FWD_ALGS>
              fwd_perf_stat;
          int returned_algo_count;
          auto cudnn_find_func = [&](void* cudnn_workspace) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                phi::dynload::cudnnFindConvolutionForwardAlgorithmEx(
                    handle,
                    x_desc,
                    transformed_input.data(),
                    w_desc,
                    filter.data(),
                    cudnn_conv_desc,
                    o_desc,
                    output->data(),
                    phi::kNUM_CUDNN_FWD_ALGS,
                    &returned_algo_count,
                    fwd_perf_stat.data(),
                    cudnn_workspace,
                    workspace_size_limit));
          };
          workspace_handle.RunFuncSync(cudnn_find_func, workspace_size_limit);
          *cudnn_algo = fwd_perf_stat[0].algo;

          PADDLE_ENFORCE_GPU_SUCCESS(
              phi::dynload::cudnnGetConvolutionForwardWorkspaceSize(
                  handle,
                  x_desc,
                  w_desc,
                  cudnn_conv_desc,
                  o_desc,
                  fwd_perf_stat[0].algo,
                  wks_bytes));
        }
      };

  auto cudnn_cache_info = CudnnConvDescManager::Instance()->GetCudnnCacheInfo(
      phi::vectorize<int>(transformed_input.dims()),
      phi::vectorize<int>(filter.dims()),
      b_dims,
      phi::vectorize<int>(output->dims()),
      conv_attr_cache->paddings,
      strides,
      conv_attr_cache->dilations,
      transformed_input.dtype(),
      groups,
      phi::backends::gpu::CudnnDataType<T>::type,
      compute_format,
      search_func,
      activation);

  auto x_desc = cudnn_cache_info->x_desc.desc();
  auto w_desc = cudnn_cache_info->w_desc.desc();
  auto b_desc = cudnn_cache_info->b_desc.desc();
  auto o_desc = cudnn_cache_info->o_desc.desc();
  auto cudnn_conv_desc = cudnn_cache_info->conv_desc.desc();
  auto act_desc = cudnn_cache_info->act_desc.desc();
  auto algo = cudnn_cache_info->algo;
  auto workspace_size = cudnn_cache_info->workspace_size;

  if ((activation == "identity") && (!residual.get_ptr())) {
    // Only the CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM algo is
    // enabled with CUDNN_ACTIVATION_IDENTITY in cuDNN lib.
    // But test in some case, the speed is slower, change to use
    // cudnnConvolutionForward and cudnnAddTensor
    // ------------- cudnn conv forward and bias add ---------------------
    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    auto cudnn_func = [&](void* cudnn_workspace) {
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnConvolutionForward(handle,
                                                &alpha,
                                                x_desc,
                                                transformed_input.data(),
                                                w_desc,
                                                filter.data(),
                                                cudnn_conv_desc,
                                                algo,
                                                cudnn_workspace,
                                                workspace_size,
                                                &beta,
                                                o_desc,
                                                output->data()));
    };
    workspace_handle.RunFunc(cudnn_func, workspace_size);
    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::cudnnAddTensor(
        handle, &alpha, b_desc, bias.data(), &alpha, o_desc, output->data()));
  } else {
    // Only the CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_​PRECOMP_GEMM algo is
    // enabled with CUDNN_ACTIVATION_IDENTITY.
    if (activation == "identity") {
      algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    }

    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = residual.get_ptr() ? 1.0f : 0.0f;
    auto cudnn_func = [&](void* cudnn_workspace) {
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnConvolutionBiasActivationForward(
              handle,
              &alpha,
              x_desc,
              transformed_input.data(),
              w_desc,
              filter.data(),
              cudnn_conv_desc,
              algo,
              cudnn_workspace,
              workspace_size,
              &beta,
              o_desc,
              residual.get_ptr() ? residual->data() : output->data(),
              b_desc,
              bias.data(),
              act_desc,
              o_desc,
              output->data()));
    };
    workspace_handle.RunFunc(cudnn_func, workspace_size);
  }

  if (!channels.empty()) {
    if (transformed_input.dims()[0] == 1 &&
        compute_format == CUDNN_TENSOR_NCHW) {
      // share data with Output
      phi::DenseTensor t;
      t.ShareDataWith(*output);
      auto y_dims = output->dims();
      t.Resize({y_dims[1], y_dims[2], y_dims[3]});
      int s = 0;
      for (size_t i = 0; i < channels.size(); ++i) {
        int e = s + channels[i];
        outs[i]->ShareDataWith(t.Slice(s, e));
        outs[i]->Resize(
            {transformed_input.dims()[0], channels[i], y_dims[2], y_dims[3]});
        s = e;
      }
    } else {
      // TODO(qingiqng): do copy when batch size large than 1
      PADDLE_THROW(phi::errors::Unimplemented(
          "Input with batch size greater than 1 is unsupported. The received "
          "batch size is %d, Input's shape is [%s].",
          transformed_input.dims()[0],
          transformed_input.dims()));
    }
  }
}
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(conv2d_fusion,  // cuda_only
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::fusion::ConvFusionKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#endif
