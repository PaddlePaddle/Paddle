
#include <hip/hip_runtime.h>
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_CUDA
#include <xxhash.h>

#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <limits>
#include <memory>
#include <unordered_map>

#include "glog/logging.h"

#include "paddle/phi/backends/context_pool.h"
#include "paddle/phi/backends/dynload/cudnn.h"
#include "paddle/phi/backends/gpu/cuda/cudnn_desc.h"
#include "paddle/phi/common/backend.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/kernels/impl/conv_cudnn_impl.h"
#include "paddle/utils/optional.h"

namespace phi {
namespace fusion {

namespace {
// TODO(wilber): Add a LRU strategy.
class CudnnConvDescManager {
 public:
  static CudnnConvDescManager* Instance() {
    static CudnnConvDescManager global;
    return &global;
  }

  struct CudnnCacheInfo {
    phi::backends::gpu::TensorDescriptor* x_desc{nullptr};
    phi::backends::gpu::FilterDescriptor* w_desc{nullptr};
    phi::backends::gpu::TensorDescriptor* b_desc{nullptr};
    phi::backends::gpu::TensorDescriptor* o_desc{nullptr};
    phi::backends::gpu::ConvolutionDescriptor* conv_desc{nullptr};
    phi::backends::gpu::ActivationDescriptor* act_desc{nullptr};
    size_t workspace_size;
    cudnnConvolutionFwdAlgo_t algo;

    std::vector<int> paddings;
    std::vector<int> dilations;
    std::vector<int> input_pad;
    std::vector<int> new_input_shape_vec;
    bool is_sys_pad;

    // TODO(wilber): The destruction of cudnn descriptor depends on the
    // phi::dynload::cudnn singleton, but when the process exits, the singleton
    // destruction order cannot be determined.
    // After testing, it is found that the phi::dynload::cudnn related singleton
    // on Windows is destructed first, causing the descriptor to be destructed
    // and failed, while the descriptor on Linux is destructed first, and the
    // phi::dynload::cudnn singleton is destructed later, so that it is correct.
    // To circumvent this problem, we rely entirely on freeing resources when
    // the process exits.

    // ~CudnnCacheInfo() {
    //   if (x_desc) delete x_desc;
    //   if (w_desc) delete w_desc;
    //   if (b_desc) delete b_desc;
    //   if (o_desc) delete o_desc;
    //   if (conv_desc) delete conv_desc;
    //   if (act_desc) delete act_desc;
    // }
  };

  CudnnCacheInfo* GetCudnnCacheInfo(
      const std::vector<int>& input_dims,
      const std::vector<int>& filter_dims,
      const std::vector<int>& bias_dims,
      const std::vector<int>& output_dims,
      const std::vector<int>& paddings,
      const std::vector<int>& strides,
      const std::vector<int>& dilations,
      phi::DataType input_dtype,
      int groups,
      cudnnDataType_t dtype,
      cudnnTensorFormat_t format,
      const std::function<void(cudnnConvolutionFwdAlgo_t*,
                               size_t*,
                               cudnnTensorDescriptor_t,
                               cudnnFilterDescriptor_t,
                               cudnnTensorDescriptor_t,
                               cudnnConvolutionDescriptor_t)>& search_func,
      const std::string& act,
      double value_max = std::numeric_limits<double>::max()) {
    // std::hash takes about 5us, xxhash can optimize to 2.5us.
    XXH64_state_t* const state = XXH64_createState();
    if (state == nullptr) {
      PADDLE_THROW(phi::errors::PreconditionNotMet(
          "xxhash create state failed, maybe a environment error."));
    }
    XXH64_hash_t const seed = 0;
    if (XXH64_reset(state, seed) == XXH_ERROR) {
      PADDLE_THROW(phi::errors::PreconditionNotMet(
          "xxhash reset state failed, maybe a environment error."));
    }
    XXH64_update(state, input_dims.data(), input_dims.size() * sizeof(int));
    XXH64_update(state, filter_dims.data(), filter_dims.size() * sizeof(int));
    XXH64_update(state, bias_dims.data(), bias_dims.size() * sizeof(int));
    // XXH64_update(state, output_dims.data(), output_dims.size() *
    // sizeof(int));
    XXH64_update(state, paddings.data(), paddings.size() * sizeof(int));
    XXH64_update(state, strides.data(), strides.size() * sizeof(int));
    XXH64_update(state, dilations.data(), dilations.size() * sizeof(int));
    XXH64_update(state, &input_dtype, sizeof(int));
    XXH64_update(state, &groups, sizeof(int));
    XXH64_update(state, &dtype, sizeof(int));
    XXH64_update(state, &format, sizeof(int));
    XXH64_update(state, act.data(), act.length() * sizeof(char));
    // XXH64_update(state, &value_max, sizeof(double));
    XXH64_hash_t hash_key = XXH64_digest(state);
    XXH64_freeState(state);

    std::lock_guard<std::mutex> lock(cache_mutex_);
    if (!cudnn_conv_cache_.count(hash_key)) {
      cudnn_conv_cache_[hash_key] = CudnnCacheInfo();
      cudnn_conv_cache_[hash_key].x_desc =
          GetTensorDescInfo(input_dims, input_dtype, format);
      cudnn_conv_cache_[hash_key].w_desc =
          GetFilterDescInfo(filter_dims, input_dtype, format);
      cudnn_conv_cache_[hash_key].o_desc =
          GetTensorDescInfo(output_dims, input_dtype, format);
      cudnn_conv_cache_[hash_key].b_desc =
          GetTensorDescInfo(bias_dims, input_dtype, format);
      cudnn_conv_cache_[hash_key].conv_desc =
          GetConvDescInfo(paddings, strides, dilations, groups, dtype);
      cudnn_conv_cache_[hash_key].act_desc =
          GetActivationDescInfo(act, value_max);

      size_t workspace_size;
      cudnnConvolutionFwdAlgo_t algo;
      search_func(&algo,
                  &workspace_size,
                  cudnn_conv_cache_[hash_key].x_desc->desc(),
                  cudnn_conv_cache_[hash_key].w_desc->desc(),
                  cudnn_conv_cache_[hash_key].o_desc->desc(),
                  cudnn_conv_cache_[hash_key].conv_desc->desc());
      cudnn_conv_cache_[hash_key].workspace_size = workspace_size;
      cudnn_conv_cache_[hash_key].algo = algo;
    }

    return &cudnn_conv_cache_.at(hash_key);
  }

  struct ConvAttrCacheInfo {
    std::vector<int> paddings;
    std::vector<int> dilations;
    std::vector<int> input_pad;
    std::vector<int> new_input_shape_vec;
    bool is_sys_pad;
  };
  ConvAttrCacheInfo* GetConvAttr(const std::vector<int>& paddings_t,
                                 const std::vector<int>& dilations_t,
                                 const std::string& padding_algorithm,
                                 const std::vector<int>& input_dims,
                                 const std::vector<int>& filter_dims,
                                 const std::vector<int>& strides,
                                 cudnnTensorFormat_t format) {
    XXH64_state_t* const state = XXH64_createState();
    if (state == nullptr) {
      PADDLE_THROW(phi::errors::PreconditionNotMet(
          "xxhash create state failed, maybe a environment error."));
    }
    XXH64_hash_t const seed = 0;
    if (XXH64_reset(state, seed) == XXH_ERROR) {
      PADDLE_THROW(phi::errors::PreconditionNotMet(
          "xxhash create state failed, maybe a environment error."));
    }
    XXH64_update(state, paddings_t.data(), paddings_t.size() * sizeof(int));
    XXH64_update(state, dilations_t.data(), dilations_t.size() * sizeof(int));
    XXH64_update(state, input_dims.data(), input_dims.size() * sizeof(int));
    XXH64_update(state, filter_dims.data(), filter_dims.size() * sizeof(int));
    XXH64_update(state, strides.data(), strides.size() * sizeof(int));
    XXH64_update(state, &format, sizeof(int));
    XXH64_update(state,
                 padding_algorithm.data(),
                 padding_algorithm.length() * sizeof(char));
    XXH64_hash_t hash_key = XXH64_digest(state);
    XXH64_freeState(state);

    std::lock_guard<std::mutex> lock(attr_mutex_);
    if (!conv_attr_cache_.count(hash_key)) {
      ConvAttrCacheInfo cache;
      auto paddings = paddings_t;
      auto dilations = dilations_t;
      std::vector<int> in_data_dims(input_dims.size() - 2);
      std::vector<int> ksize(filter_dims.size() - 2);
      if (format == CUDNN_TENSOR_NHWC) {
        for (size_t i = 1; i < input_dims.size() - 1; ++i) {
          in_data_dims[i - 1] = input_dims[i];
        }
        for (size_t i = 1; i < filter_dims.size() - 1; ++i) {
          ksize[i - 1] = filter_dims[i];
        }
      } else {
        for (size_t i = 2; i < input_dims.size(); ++i) {
          in_data_dims[i - 2] = input_dims[i];
        }
        for (size_t i = 2; i < filter_dims.size(); ++i) {
          ksize[i - 2] = filter_dims[i];
        }
      }
      phi::UpdatePaddingAndDilation(&paddings,
                                    &dilations,
                                    padding_algorithm,
                                    make_ddim(in_data_dims),
                                    strides,
                                    ksize);

      int data_dim = strides.size();  // 2d or 3d
      bool is_sys_pad = funcs::IsSymmetricPadding(paddings, data_dim);
      std::vector<int> padding_common(data_dim, 0);
      if (!is_sys_pad) {
        std::vector<int> padding_diff(data_dim);
        std::vector<int> new_input_shape_vec(data_dim + 2);
        new_input_shape_vec[0] = input_dims[0];

        if (format == CUDNN_TENSOR_NCHW) {
          new_input_shape_vec[1] = input_dims[1];
        } else {
          new_input_shape_vec[data_dim + 1] = input_dims[data_dim + 1];
        }

        std::vector<int> input_pad(input_dims.size() * 2, 0);
        for (size_t i = 0; i < data_dim; ++i) {
          padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
          padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
          if (format == CUDNN_TENSOR_NCHW) {
            new_input_shape_vec[i + 2] = input_dims[i + 2] + padding_diff[i];
          } else {
            new_input_shape_vec[i + 1] = input_dims[i + 1] + padding_diff[i];
          }
          if (format == CUDNN_TENSOR_NCHW) {
            input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
            input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
          } else {
            input_pad[2 * i + 2] = paddings[2 * i] - padding_common[i];
            input_pad[2 * i + 2 + 1] = paddings[2 * i + 1] - padding_common[i];
          }
        }

        cache.is_sys_pad = false;
        cache.input_pad = input_pad;
        cache.new_input_shape_vec = new_input_shape_vec;
      } else {
        cache.is_sys_pad = true;
        if (paddings.size() == data_dim) {
          for (size_t i = 0; i < data_dim; ++i) {
            padding_common[i] = paddings[i];
          }
        } else {
          for (size_t i = 0; i < data_dim; ++i) {
            padding_common[i] = paddings[2 * i];
          }
        }
      }

      cache.dilations = dilations;
      cache.paddings = padding_common;
      conv_attr_cache_[hash_key] = cache;
    }

    return &conv_attr_cache_.at(hash_key);
  }

 private:
  phi::backends::gpu::TensorDescriptor* GetTensorDescInfo(
      const std::vector<int>& input_dims,
      phi::DataType input_dtype,
      cudnnTensorFormat_t input_format) {
    auto* desc = new phi::backends::gpu::TensorDescriptor();
    desc->set(
        input_dims, input_format, backends::gpu::ToCudnnDataType(input_dtype));
    return desc;
  }

  phi::backends::gpu::FilterDescriptor* GetFilterDescInfo(
      const std::vector<int>& input_dims,
      phi::DataType input_dtype,
      cudnnTensorFormat_t input_format) {
    auto* desc = new phi::backends::gpu::FilterDescriptor();
    desc->set(
        input_dims, input_format, backends::gpu::ToCudnnDataType(input_dtype));
    return desc;
  }

  phi::backends::gpu::ConvolutionDescriptor* GetConvDescInfo(
      const std::vector<int>& paddings,
      const std::vector<int>& strides,
      const std::vector<int>& dilations,
      int groups,
      cudnnDataType_t dtype) {
    auto* desc = new phi::backends::gpu::ConvolutionDescriptor();
    desc->set(
        dtype, paddings, strides, dilations, phi::AllowTF32Cudnn(), groups);
    return desc;
  }

  phi::backends::gpu::ActivationDescriptor* GetActivationDescInfo(
      const std::string& act,
      double value_max = std::numeric_limits<double>::max()) {
    auto* desc = new phi::backends::gpu::ActivationDescriptor();
    cudnnActivationMode_t mode;
    double relu_ceiling = 0.0;
    if (act == "identity") {
      mode = CUDNN_ACTIVATION_IDENTITY;
    } else if (act == "relu") {
      mode = CUDNN_ACTIVATION_RELU;
    } else if (act == "relu6") {
      relu_ceiling = 6.0;
      mode = CUDNN_ACTIVATION_CLIPPED_RELU;
    } else if (act == "sigmoid") {
      mode = CUDNN_ACTIVATION_SIGMOID;
    } else if (act == "relux") {
      relu_ceiling = value_max;
      mode = CUDNN_ACTIVATION_CLIPPED_RELU;
    } else if (act == "tanh") {
      mode = CUDNN_ACTIVATION_TANH;
    } else {
      PADDLE_THROW(phi::errors::Unimplemented(
          "Unknown CUDNN activation string: %s.", act));
    }
    desc->set(mode, relu_ceiling);
    return desc;
  }

  std::mutex cache_mutex_;
  std::unordered_map<size_t, CudnnCacheInfo> cudnn_conv_cache_;

  std::mutex attr_mutex_;
  std::unordered_map<size_t, ConvAttrCacheInfo> conv_attr_cache_;
};
}  // namespace

template <typename T, typename Context>
void ConvFusionKernel(const Context& ctx,
                      const DenseTensor& input,
                      const DenseTensor& filter,
                      const DenseTensor& bias,
                      const paddle::optional<DenseTensor>& residual,
                      const std::vector<int>& strides,
                      const std::vector<int>& paddings_t,
                      const std::string& padding_algorithm,
                      const std::vector<int>& dilations_t,
                      int groups,
                      const std::string& data_format,
                      const std::string& activation,
                      bool exhaustive_search,
                      const std::vector<int>& channels,
                      int user_workspace_size,
                      DenseTensor* output,
                      std::vector<DenseTensor*> outs) {
  auto handle = ctx.cudnn_handle();
  ctx.template Alloc<T>(output);
  auto workspace_handle = ctx.cudnn_workspace_handle();

  exhaustive_search = FLAGS_cudnn_exhaustive_search || exhaustive_search;
  bool deterministic = FLAGS_cudnn_deterministic;
  PADDLE_ENFORCE_EQ(exhaustive_search && deterministic,
                    false,
                    phi::errors::InvalidArgument(
                        "Cann't set exhaustive_search True and "
                        "FLAGS_cudnn_deterministic True at same time."));

  size_t workspace_size_limit = 0;
  if (FLAGS_conv_workspace_size_limit > 0 || user_workspace_size > 0) {
    int64_t max_user_size =
        std::min(static_cast<int64_t>(FLAGS_conv_workspace_size_limit),
                 static_cast<int64_t>(user_workspace_size));
    workspace_size_limit = max_user_size * 1024 * 1024;
  }

  auto dtype = phi::backends::gpu::CudnnDataType<T>::type;
  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");
  // Choose NHWC or NCHW by data_format attr.
  auto compute_format = channel_last ? CUDNN_TENSOR_NHWC : CUDNN_TENSOR_NCHW;
  VLOG(3) << "Compute ConvFusionOp with cuDNN:"
          << " data_format=" << data_format << " compute_format="
          << (compute_format == CUDNN_TENSOR_NHWC ? "NHWC" : "NCHW");

  auto* conv_attr_cache = CudnnConvDescManager::Instance()->GetConvAttr(
      paddings_t,
      dilations_t,
      padding_algorithm,
      phi::vectorize<int>(input.dims()),
      phi::vectorize<int>(filter.dims()),
      strides,
      compute_format);

  DenseTensor transformed_input;
  const int input_rank = input.dims().size();
  auto unsys_pad_process = [&](const std::vector<int>& new_input_shape_vec,
                               const std::vector<int>& input_pad) {
    DDim new_input_shape(make_ddim(new_input_shape_vec));
    transformed_input.Resize(new_input_shape);
    ctx.template Alloc<T>(&transformed_input);

    T pad_value(0.0);
    switch (input_rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(
            ctx, input_pad, input, pad_value, &transformed_input);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(
            ctx, input_pad, input, pad_value, &transformed_input);
      } break;
      default:
        PADDLE_THROW(phi::errors::InvalidArgument(
            "ConvOp only support tensors with 4 or 5 dimensions."));
    }
  };
  if (conv_attr_cache->is_sys_pad) {
    transformed_input.ShareDataWith(input);
  } else {
    unsys_pad_process(conv_attr_cache->new_input_shape_vec,
                      conv_attr_cache->input_pad);
  }

  std::vector<int> b_dims(input_rank, 1);
  if (compute_format == CUDNN_TENSOR_NCHW) {
    auto bias_rank = bias.dims().size();
    if (input_rank == bias_rank) {
      b_dims[1] = static_cast<int>(bias.dims()[1]);
    } else {
      b_dims[1] = static_cast<int>(bias.dims()[0]);
    }
  } else {
    b_dims[input_rank - 1] = static_cast<int>(bias.dims()[0]);
  }

  auto search_func = [&](cudnnConvolutionFwdAlgo_t* cudnn_algo,
                         size_t* wks_bytes,
                         cudnnTensorDescriptor_t x_desc,
                         cudnnFilterDescriptor_t w_desc,
                         cudnnTensorDescriptor_t o_desc,
                         cudnnConvolutionDescriptor_t cudnn_conv_desc) {
    if (!exhaustive_search) {
#if CUDNN_VERSION >= 8000
      int perf_count;
      int best_algo_idx = 0;
      size_t tmp_size = 0;
      std::unique_ptr<cudnnConvolutionFwdAlgoPerf_t[]> perf_results(
          new cudnnConvolutionFwdAlgoPerf_t[phi::kNUM_CUDNN_FWD_ALGS]);
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnGetConvolutionForwardAlgorithm_v7(
              handle,
              x_desc,
              w_desc,
              cudnn_conv_desc,
              o_desc,
              phi::kNUM_CUDNN_FWD_ALGS,
              &perf_count,
              perf_results.get()));
      *cudnn_algo = (perf_results.get())[best_algo_idx].algo;
#else
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnGetConvolutionForwardAlgorithm(
              handle,
              x_desc,
              w_desc,
              cudnn_conv_desc,
              o_desc,
              CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
              workspace_size_limit,
              cudnn_algo));
#endif
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnGetConvolutionForwardWorkspaceSize(handle,
                                                                x_desc,
                                                                w_desc,
                                                                cudnn_conv_desc,
                                                                o_desc,
                                                                *cudnn_algo,
                                                                wks_bytes));
    } else {
      std::array<cudnnConvolutionFwdAlgoPerf_t, phi::kNUM_CUDNN_FWD_ALGS>
          fwd_perf_stat;
      int returned_algo_count;
      auto cudnn_find_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_GPU_SUCCESS(
            phi::dynload::cudnnFindConvolutionForwardAlgorithmEx(
                handle,
                x_desc,
                transformed_input.data(),
                w_desc,
                filter.data(),
                cudnn_conv_desc,
                o_desc,
                output->data(),
                phi::kNUM_CUDNN_FWD_ALGS,
                &returned_algo_count,
                fwd_perf_stat.data(),
                cudnn_workspace,
                workspace_size_limit));
      };
      workspace_handle.RunFuncSync(cudnn_find_func, workspace_size_limit);
      *cudnn_algo = fwd_perf_stat[0].algo;

      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnGetConvolutionForwardWorkspaceSize(
              handle,
              x_desc,
              w_desc,
              cudnn_conv_desc,
              o_desc,
              fwd_perf_stat[0].algo,
              wks_bytes));
    }
  };

  auto cudnn_cache_info = CudnnConvDescManager::Instance()->GetCudnnCacheInfo(
      phi::vectorize<int>(transformed_input.dims()),
      phi::vectorize<int>(filter.dims()),
      b_dims,
      phi::vectorize<int>(output->dims()),
      conv_attr_cache->paddings,
      strides,
      conv_attr_cache->dilations,
      transformed_input.dtype(),
      groups,
      phi::backends::gpu::CudnnDataType<T>::type,
      compute_format,
      search_func,
      activation);

  auto x_desc = cudnn_cache_info->x_desc->desc();
  auto w_desc = cudnn_cache_info->w_desc->desc();
  auto b_desc = cudnn_cache_info->b_desc->desc();
  auto o_desc = cudnn_cache_info->o_desc->desc();
  auto cudnn_conv_desc = cudnn_cache_info->conv_desc->desc();
  auto act_desc = cudnn_cache_info->act_desc->desc();
  auto algo = cudnn_cache_info->algo;
  auto workspace_size = cudnn_cache_info->workspace_size;

  if ((activation == "identity") && (!residual.get_ptr())) {
    // Only the CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM algo is
    // enabled with CUDNN_ACTIVATION_IDENTITY in cuDNN lib.
    // But test in some case, the speed is slower, change to use
    // cudnnConvolutionForward and cudnnAddTensor
    // ------------- cudnn conv forward and bias add ---------------------
    ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
    auto cudnn_func = [&](void* cudnn_workspace) {
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnConvolutionForward(handle,
                                                &alpha,
                                                x_desc,
                                                transformed_input.data(),
                                                w_desc,
                                                filter.data(),
                                                cudnn_conv_desc,
                                                algo,
                                                cudnn_workspace,
                                                workspace_size,
                                                &beta,
                                                o_desc,
                                                output->data()));
    };
    workspace_handle.RunFunc(cudnn_func, workspace_size);
    PADDLE_ENFORCE_GPU_SUCCESS(phi::dynload::cudnnAddTensor(
        handle, &alpha, b_desc, bias.data(), &alpha, o_desc, output->data()));
  } else {
    // Only the CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_​PRECOMP_GEMM algo is
    // enabled with CUDNN_ACTIVATION_IDENTITY.
    if (activation == "identity") {
      algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    }

    ScalingParamType<T> alpha = 1.0f;
    ScalingParamType<T> beta = residual.get_ptr() ? 1.0f : 0.0f;
    auto cudnn_func = [&](void* cudnn_workspace) {
      PADDLE_ENFORCE_GPU_SUCCESS(
          phi::dynload::cudnnConvolutionBiasActivationForward(
              handle,
              &alpha,
              x_desc,
              transformed_input.data(),
              w_desc,
              filter.data(),
              cudnn_conv_desc,
              algo,
              cudnn_workspace,
              workspace_size,
              &beta,
              o_desc,
              residual.get_ptr() ? residual->data() : output->data(),
              b_desc,
              bias.data(),
              act_desc,
              o_desc,
              output->data()));
    };
    workspace_handle.RunFunc(cudnn_func, workspace_size);
  }

  if (!channels.empty()) {
    if (transformed_input.dims()[0] == 1 &&
        compute_format == CUDNN_TENSOR_NCHW) {
      // share data with Output
      phi::DenseTensor t;
      t.ShareDataWith(*output);
      auto y_dims = output->dims();
      t.Resize({y_dims[1], y_dims[2], y_dims[3]});
      int s = 0;
      for (size_t i = 0; i < channels.size(); ++i) {
        int e = s + channels[i];
        outs[i]->ShareDataWith(t.Slice(s, e));
        outs[i]->Resize(
            {transformed_input.dims()[0], channels[i], y_dims[2], y_dims[3]});
        s = e;
      }
    } else {
      // TODO(qingiqng): do copy when batch size large than 1
      PADDLE_THROW(phi::errors::Unimplemented(
          "Input with batch size greater than 1 is unsupported. The received "
          "batch size is %d, Input's shape is [%s].",
          transformed_input.dims()[0],
          transformed_input.dims()));
    }
  }
}
}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(conv2d_fusion,  // cuda_only
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::fusion::ConvFusionKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#endif
