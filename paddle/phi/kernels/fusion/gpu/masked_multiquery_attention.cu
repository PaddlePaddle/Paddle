#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once

#include "paddle/phi/kernels/fusion/gpu/masked_multiquery_attention.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {
namespace fusion {

template <typename T, typename Context>
void MMQAKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const paddle::optional<DenseTensor>& kv_input,
                const paddle::optional<DenseTensor>& bias,
                const paddle::optional<DenseTensor>& src_mask,
                const paddle::optional<DenseTensor>& sequence_lengths,
                const paddle::optional<DenseTensor>& rotary_tensor,
                const paddle::optional<DenseTensor>& beam_cache_offset,
                const DenseTensor& cache_kv,
                const paddle::optional<DenseTensor>& qkv_out_scale,
                const paddle::optional<DenseTensor>& out_linear_shift,
                const paddle::optional<DenseTensor>& out_linear_smooth,
                int beam_size,
                int rotary_emb_dims,
                const bool split_kv,
                const int head_kv,
                const bool mask_broadcast_num_heads,
                const bool compute_bias,
                const bool use_neox_rotary_style,
                const float out_linear_in_scale,
                const int quant_round_type,
                const float quant_max_bound,
                const float quant_min_bound,
                DenseTensor* out,
                DenseTensor* cache_kv_out,
                DenseTensor* beam_cache_offset_out) {
  Masked_multiquery_attention_params<T> params;
  const auto& x_dims = x.dims();
  int bsz = x_dims[0];
  int num_head = 0;
  if (split_kv) {
    num_head = x_dims[1];
  } else {
    num_head = x_dims[1] - head_kv * 2;
  }

  int dim_head = x_dims[2];
  int timestep = src_mask->dims()[3] - 1;
  int cache_bsz = cache_kv.dims()[1];
  int max_seq_len = cache_kv.dims()[3];
  float inv_sqrt_dh = 1. / sqrt(dim_head);

  if (out_linear_in_scale > 0) {
    dev_ctx.template Alloc<int8_t>(out);
  } else {
    dev_ctx.template Alloc<T>(out);
  }
  params.attn_mask = src_mask->data<T>();
  params.mask_broadcast_num_heads = mask_broadcast_num_heads;
  params.cache_kv = const_cast<T*>(cache_kv_out->data<T>());
  params.neox_rotary_style = use_neox_rotary_style;
  params.mask_length = src_mask->dims()[3];

  // params.mqa = mqa;
  if (sequence_lengths) {
    params.sequence_lengths = sequence_lengths->data<int>();
  }
  if (rotary_emb_dims > 0) {
    params.rotary_emb = rotary_tensor->data<float>();
  } else {
    params.rotary_emb = nullptr;
  }

  if (beam_cache_offset) {
    params.beam_cache_offset = beam_cache_offset->data<int>();
  }

  params.add_qkv_bias = compute_bias;
  if (compute_bias) {
    // Because we may not add qkv_bias, so here we cast to T*.
    // Author(zhengzekang).
    params.qkv_bias = const_cast<T*>(bias->data<T>());
  }

  params.batch_size = bsz;
  params.cache_batch_size = cache_bsz;
  params.beam_width = beam_size;
  params.num_head = num_head;
  params.timestep = timestep;
  params.max_seq_length = max_seq_len;
  params.inv_sqrt_dh = inv_sqrt_dh;
  params.rotary_emb_dims = rotary_emb_dims;
  params.head_kv = head_kv;
  params.split_kv = split_kv;
  if (out_linear_shift) {
    DispatchFMQA<T>(dev_ctx,
                    x,
                    *(out_linear_shift.get_ptr()),
                    *(out_linear_smooth.get_ptr()),
                    params,
                    num_head,
                    dim_head,
                    out,
                    kv_input.get_ptr(),
                    qkv_out_scale.get_ptr(),
                    out_linear_in_scale,
                    quant_round_type,
                    quant_max_bound,
                    quant_min_bound);
  } else {
    DispatchFMQA<T>(dev_ctx,
                    x,
                    params,
                    num_head,
                    dim_head,
                    out,
                    kv_input.get_ptr(),
                    qkv_out_scale.get_ptr(),
                    out_linear_in_scale,
                    quant_round_type,
                    quant_max_bound,
                    quant_min_bound);
  }
}

}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(masked_multiquery_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::MMQAKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
