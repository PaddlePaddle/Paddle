/* Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <array>

#include "paddle/common/flags.h"
#include "paddle/phi/backends/gpu/gpu_dnn.h"
#include "paddle/phi/backends/gpu/gpu_info.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/autotune/cache.h"
#include "paddle/phi/kernels/funcs/pooling.h"
#include "paddle/phi/kernels/gpudnn/conv_cudnn_frontend.h"
#include "paddle/phi/kernels/gpudnn/pool_gpudnn.h"

COMMON_DECLARE_bool(cudnn_exhaustive_search);

namespace phi {

template <typename Context, typename T1, typename T2 = int>
void MaxPoolV2CUDNNKernel(const Context& ctx,
                          const DenseTensor& x,
                          const std::vector<int>& kernel_size,
                          const std::vector<int>& strides,
                          const std::vector<int>& paddings,
                          const std::string& data_format,
                          bool global_pooling,
                          bool adaptive,
                          DenseTensor* out,
                          DenseTensor* saved_idx) {
  PADDLE_ENFORCE_GE(ctx.GetComputeCapability(),
                    80,
                    phi::errors::PreconditionNotMet(
                        "This op only supports Ampere and later devices, "
                        "but got compute capability: %d.",
                        ctx.GetComputeCapability()));
  // Additional options
  bool exhaustive_search = FLAGS_cudnn_exhaustive_search;
  bool deterministic = FLAGS_cudnn_deterministic;
  PADDLE_ENFORCE_EQ(exhaustive_search && deterministic,
                    false,
                    phi::errors::InvalidArgument(
                        "Cann't set exhaustive_search True and "
                        "FLAGS_cudnn_deterministic True at same time."));
  // Allocate output tensors
  ctx.template Alloc<T1>(out);
  ctx.template Alloc<T2>(saved_idx);
  // Update paddings
  std::vector<int> paddings_ = paddings;
  std::vector<int> kernel_size_ = kernel_size;
  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");
  const std::string padding_algorithm = "EXPLICIT";

  auto x_dims = x.dims();
  DDim data_dims;
  if (channel_last) {
    data_dims = slice_ddim(x_dims, 1, x_dims.size() - 1);
  } else {
    data_dims = slice_ddim(x_dims, 2, x_dims.size());
  }
  funcs::UpdatePadding(&paddings_,
                       global_pooling,
                       adaptive,
                       padding_algorithm,
                       data_dims,
                       strides,
                       kernel_size_);

  const auto data_dim = data_dims.size();
  std::vector<int64_t> pre_padding(data_dim, 0);
  std::vector<int64_t> post_padding(data_dim, 0);
  for (size_t i = 0; i < data_dim; ++i) {
    pre_padding[i] = static_cast<int64_t>(paddings_[2 * i]);
    post_padding[i] = static_cast<int64_t>(paddings_[2 * i + 1]);
  }

  if (global_pooling) {
    funcs::UpdateKernelSize(&kernel_size_, data_dims);
  }

  using helper = CudnnFrontendConvHelper;
  auto kernel_size_int64 = helper::GetInt64Array(kernel_size_);
  auto strides_int64 = helper::GetInt64Array(strides);

  // Prepare for execution
  auto& plan_cache = phi::autotune::AutoTuneCache::Instance().GetConvV8(
      phi::autotune::AlgorithmType::kPoolingForwardV8);

  T1* input_data = const_cast<T1*>(x.data<T1>());
  T1* output_data = out->data<T1>();
  T2* saved_idx_data = saved_idx->data<T2>();

  hipdnnHandle_t handle = const_cast<hipdnnHandle_t>(ctx.cudnn_handle());
  auto workspace_handle = ctx.cudnn_workspace_handle();

  auto layout = GetLayoutFromStr(data_format);
  auto layout_format = phi::backends::gpu::GetCudnnTensorFormat(layout);
  auto input_dtype = phi::backends::gpu::CudnnDataType<T1>::type;
  auto saved_idx_dtype = CudnnIndexType<T2>::type;

  // Create plan and execute
  std::vector<void*> data_ptrs({input_data, output_data, saved_idx_data});
  std::vector<int64_t> uids({'x', 'o', 's'});

  // Create feature vector for plan caching
  cudnn_frontend::feature_vector_t feature_vector;
  auto dim_x = phi::vectorize<int64_t>(x.dims());

  phi::autotune::BuildFeatureVector(&feature_vector,
                                    dim_x,
                                    kernel_size_int64,
                                    strides_int64,
                                    pre_padding,
                                    post_padding,
                                    data_format,
                                    input_dtype,
                                    saved_idx_dtype);

  // Query cache and execute
  if (plan_cache.FindPlan(feature_vector, handle)) {
    const cudnn_frontend::ExecutionPlan* cached_plan = nullptr;
    int64_t workspace_size = 0;
    plan_cache.GetPlanAndWorkspaceSize(
        feature_vector, &cached_plan, &workspace_size, handle);
    helper::ExecutePlan(handle,
                        &workspace_handle,
                        &data_ptrs,
                        &uids,
                        cached_plan->get_raw_desc(),
                        workspace_size);
    return;
  }

  // Create tensor descriptors
  auto x_desc = helper::GetTensorDescriptor(&x, 'x', layout_format);
  auto out_desc = helper::GetTensorDescriptor(out, 'o', layout_format);
  auto saved_idx_desc =
      helper::GetTensorDescriptor(saved_idx, 's', layout_format);

  // Create maxpooling descriptor
  auto const nan_opt = HIPDNN_NOT_PROPAGATE_NAN;
  auto const mode = cudnn_frontend::cudnnResampleMode_t::CUDNN_RESAMPLE_MAXPOOL;
  auto const padding_mode =
      cudnn_frontend::cudnnPaddingMode_t::CUDNN_NEG_INF_PAD;
  auto pool_desc = cudnn_frontend::ResampleDescBuilder_v8()
                       .setComputeType(HIPDNN_DATA_FLOAT)
                       .setNanPropagation(nan_opt)
                       .setResampleMode(mode)
                       .setPaddingMode(padding_mode)
                       .setSpatialDim(data_dim, kernel_size_int64.data())
                       .setSpatialStride(data_dim, strides_int64.data())
                       .setPrePadding(data_dim, pre_padding.data())
                       .setPostPadding(data_dim, post_padding.data())
                       .build();

  // Create maxpooling op
  auto pool_op = cudnn_frontend::OperationBuilder(
                     CUDNN_BACKEND_OPERATION_RESAMPLE_FWD_DESCRIPTOR)
                     .setxDesc(x_desc)
                     .setyDesc(out_desc)
                     .setidxDesc(saved_idx_desc)
                     .setResampleDesc(pool_desc)
                     .build();

  // Create op graph
  std::array<cudnn_frontend::Operation const*, 1> ops = {&pool_op};
  auto op_graph = cudnn_frontend::OperationGraphBuilder()
                      .setHandle(handle)
                      .setOperationGraph(ops.size(), ops.data())
                      .build();

  auto plans = helper::FindExecutionPlans(&op_graph,
                                          exhaustive_search,
                                          deterministic,
                                          &data_ptrs,
                                          &uids,
                                          handle,
                                          &workspace_handle);

  helper::ExecutePlansAndCache(handle,
                               &workspace_handle,
                               &data_ptrs,
                               &uids,
                               &plans,
                               exhaustive_search,
                               feature_vector,
                               &plan_cache);
}

template <typename T, typename Context>
void MaxPool2dV2CUDNNKernel(const Context& ctx,
                            const DenseTensor& x,
                            const std::vector<int>& kernel_size,
                            const std::vector<int>& strides,
                            const std::vector<int>& paddings,
                            const std::string& data_format,
                            bool global_pooling,
                            bool adaptive,
                            DenseTensor* out,
                            DenseTensor* saved_idx) {
  // TODO(tizheng): support int8 mask
  MaxPoolV2CUDNNKernel<Context, T>(ctx,
                                   x,
                                   kernel_size,
                                   strides,
                                   paddings,
                                   data_format,
                                   global_pooling,
                                   adaptive,
                                   out,
                                   saved_idx);
}

}  // namespace phi

using phi::dtype::float16;

PD_REGISTER_KERNEL(max_pool2d_v2,  // cuda_only
                   GPU,
                   ALL_LAYOUT,
                   phi::MaxPool2dV2CUDNNKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {
  kernel->OutputAt(1).SetDataType(phi::CppTypeToDataType<int>::Type());
}
