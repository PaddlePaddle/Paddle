#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/fusion/gpu/mmha_util.cu.h"

namespace phi {
namespace fusion {

#ifndef PADDLE_WITH_HIP

constexpr unsigned int str2int(const char *str, int h = 0) {
  return !str[h] ? 5381 : (str2int(str, h + 1) * 33) ^ str[h];
}

template <typename T>
struct Masked_multihead_attention_params {
  // output buffer, [B, 1(seq_len), num_head * dim_head]
  const T *q;
  const T *k;
  const T *v;
  T *out;
  // qkv_out, [B, 1(seq_len), 3, num_head * dim_head]
  const T *qkv;
  // bias, [3, num_head, dim_head]
  T *qkv_bias;
  // [bsz, seq_len]
  const int *cum_offsets;
  // TODO(wangxi): optimize with input_lengths and max_input_len?
  // [bsz, 1, 1, time_step(cache_seq_length)+1]
  const T *attn_mask;
  int mask_length;
  // whether to broadcast num_heads(2nd) dimension for attn_mask
  // in MMHA, if false, attn_mask shape should be
  // [bsz, num_heads, 1, time_step(cache_seq_length)+1]
  bool mask_broadcast_num_heads;

  // [2, B, num_head, max_seq_len(valid cache_seq_len), dim_head]
  // k [B, num_head, dim_head/x, max_seq_len, x], that is `seq_len` first
  // v [B, num_head, max_seq_len, dim_head]
  T *cache_kv;
  // [B, max_seq_len]
  const int *beam_cache_offset = nullptr;

  const int *sequence_lengths{nullptr};

  // The RoPE embedding, [2, B, rotary_seq_len, 1, dim_head]
  // rotary_emb_dims = 1 if pos_ids_extra is null else 2
  const float *rotary_emb;
  int rotary_emb_dims;
  int rotary_seq_len = 1;

  int batch_size;  // batch * beam
  int beam_width;
  int cache_batch_size;
  int num_head;
  // k_num_head and v_num_head must be equal, we unify them.
  // kv_num_head = k_num_head && kv_num_head == v_num_head
  int kv_num_head;
  int timestep;  // cache_seq_length
  int seq_len;
  int max_seq_length;

  // 1.f / sqrt(Dh)
  float inv_sqrt_dh;

  bool add_qkv_bias;
  bool neox_rotary_style;
};

template <typename T,
          int Dh,
          int Dh_MAX,
          int THREADS_PER_KEY,
          int THREADS_PER_VALUE,
          int THREADS_PER_BLOCK,
          typename LoadFunc,
          typename StoreFunc>
__global__ void qkv_attention_kernel(
    Masked_multihead_attention_params<T> params,
    LoadFunc load_func,
    StoreFunc store_func) {
  // printf("-------------\n");
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const int bi = blockIdx.y;
  // params.sequence_lengths[bi] means how many k and v we have cached in
  // cache_kv.

  // Dh = 128
  // Dh_max = 128
  // THREADS_PER_BLOCK = 128
  // THDS_PER_KEY = 2
  // THREADS_PER_VALUE = 16
  //  WARPS_PER_BLOCK = 4
  //  THREADS_PER_KEY = 2

  // if( threadIdx.x == 0 && bi == 0 && blockIdx.x == 0)
  // {
  //   printf("param %d %d %d %d %d\n", Dh, Dh_MAX, THREADS_PER_KEY,
  //   THREADS_PER_VALUE , THREADS_PER_BLOCK );
  // }

  typedef PDDataTypeTraits<T> traits_;
  typedef typename traits_::DataType DataType_;

  static_assert(Dh_MAX % THREADS_PER_KEY == 0, "");
  static_assert(Dh_MAX % THREADS_PER_VALUE == 0, "");

  constexpr int WARP_SIZE = 32;
  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  extern __shared__ char smem_[];

  float *qk_smem = reinterpret_cast<float *>(smem_);

  char *logits_smem_ = smem_;
  // fp32 accum for logits
  float *logits_smem = reinterpret_cast<float *>(logits_smem_);

  T *out_smem = reinterpret_cast<T *>(smem_);

  __shared__ float red_smem[WARPS_PER_BLOCK * 2];
  using Qk_vec = typename Qk_vec_<T, Dh_MAX>::Type;
  using Qk_vec_RoPE = typename Qk_vec_RoPE_<T, float, Dh_MAX>::Type;
  __shared__ __align__(sizeof(Qk_vec)) T q_smem[Dh_MAX];

  // beam id
  const int beami = bi % params.beam_width;
  // real batch id
  const int bbi = bi / params.beam_width;
  const int hi = blockIdx.x;
  const int bhi = bi * params.num_head + hi;

  const int kv_num_head = params.kv_num_head;
  const int num_head_per_group = params.num_head / kv_num_head;
  // hi means the head index in query processed by this cuda thread.
  // kv_bhi means the merged batch and head index in key and value processed by
  // this cuda thread.
  const int kv_bhi = bi * kv_num_head + hi / num_head_per_group;

  const int bbhi = bbi * params.beam_width * params.num_head + hi;
  const int ti =
      params.cum_offsets ? bi * params.seq_len - params.cum_offsets[bi] : -1;
  const int thi = params.cum_offsets ? ti * params.num_head + hi : -1;
  const int tid = threadIdx.x;

  const int bi_seq_len_offset = bi * params.max_seq_length;

  float qk_max = -FLT_MAX;
  float qk = 0;

  int act_time_step = params.timestep;

  // qkv [B, S=1, num_head + 2 * kv_num_head, head_dim]
  // this hi means the head index in query!
  int qkv_base_offset = bi * (params.num_head) * Dh + hi * Dh;

  // QK_VEC_SIZE == 4??
  constexpr int QK_VEC_SIZE = sizeof(Qk_vec) / sizeof(T);
  static_assert(Dh_MAX % QK_VEC_SIZE == 0, "");
  // Use block reduction if needed
  // static_assert(Dh_MAX / QK_VEC_SIZE <= WARP_SIZE, "");
  // WARPS_PER_BLOCK = 128 / 4 = 32
  constexpr int QK_VECS_PER_WARP = Dh_MAX / QK_VEC_SIZE;

  // cache_k, [B, num_head, head_dim / x, max_seq_len, x]
  // x == 4/8 for FP32/FP16, 128bit, 16Byte
  constexpr int QK_ELTS_IN_16B = 16 / sizeof(T);       // 8
  constexpr int QK_VECS_IN_16B = 16 / sizeof(Qk_vec);  // 2

  // printf("qk vec  %d %d\n", QK_VEC_SIZE, QK_VECS_PER_WARP);

  // load q element to q smem
  if (tid < QK_VECS_PER_WARP) {
    int qk_offset = qkv_base_offset + tid * QK_VEC_SIZE;
    int q_bias_offset = hi * Dh + tid * QK_VEC_SIZE;
    int k_bias_offset = hi / num_head_per_group * Dh + tid * QK_VEC_SIZE;

    Qk_vec q;
    zero(q);
    // q = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
    //         ? *reinterpret_cast<const Qk_vec *>(&q_base[qk_offset])
    //         : q;
    if (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh) {
      load_func.template load<Qk_vec>(q, qk_offset);
    }

    *reinterpret_cast<Qk_vec *>(&q_smem[tid * QK_VEC_SIZE]) = q;
  }

  __syncthreads();

  using K_vec = typename K_vec_<T, THREADS_PER_KEY>::Type;
  constexpr int K_VEC_SIZE = sizeof(K_vec) / sizeof(T);  // 4
  static_assert(Dh_MAX % K_VEC_SIZE == 0, "");
  constexpr int K_ELTS_PER_THREAD = Dh_MAX / THREADS_PER_KEY;  // 128 / 2 = 64
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;  // 16

  int ko = tid / THREADS_PER_KEY;
  int ki = (tid % THREADS_PER_KEY) * K_VEC_SIZE;

  static_assert(Dh_MAX == THREADS_PER_KEY * K_VEC_SIZE * K_VECS_PER_THREAD, "");

  // printf("k vec   %d %d %d %d\n", K_VECS_PER_THREAD, K_VEC_SIZE ,
  // K_VECS_PER_THREAD, K_ELTS_PER_THREAD);

  // bfloat4[16] , each thread read 64 ele
  K_vec q[K_VECS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < K_VECS_PER_THREAD; ++i) {
    q[i] = *reinterpret_cast<const K_vec *>(
        &q_smem[ki + i * THREADS_PER_KEY * K_VEC_SIZE]);
  }

  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;  // 128 2 = 64
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;          // ==2

  T *k_cache = &params.cache_kv[kv_bhi * params.max_seq_length * Dh + ki];
  T *k_cache_batch = &params.cache_kv[bbhi * params.max_seq_length * Dh + ki];
  int ti_end = div_up(act_time_step, K_PER_WARP) * K_PER_WARP;

  // each thread process act_time_step
  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    K_vec k[K_VECS_PER_THREAD];
    K_vec k_vec_zero;
    zero(k_vec_zero);
    // if( threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //   printf("begin \n", qk);
    // }
#pragma unroll
    for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
      int jj = ii * params.max_seq_length + ti;
      if (ti < act_time_step) {
        k[ii] = *reinterpret_cast<const K_vec *>(
            &params.k[ti * params.num_head * Dh + ki +
                      ii * THREADS_PER_KEY * K_VEC_SIZE + hi * Dh]);
      }
    }

    float qk = Qk_dot<T, THREADS_PER_KEY>::dot(q, k, params.inv_sqrt_dh);
    const T *q_ptr = reinterpret_cast<const T *>(q);
    const T *k_ptr = reinterpret_cast<const T *>(k);

    // if( threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //   printf("q k %f %f\n", float(q_ptr[0]), float(k_ptr[0]));
    // }

    // NOTE(liyurui): We should multiple q with inv_sqrt_dh first, for dot(q, k)
    // may overflow with FP16 in large model.

    // if( (threadIdx.x == 0 || threadIdx.x == 1 ) && blockIdx.x == 0 &&
    // blockIdx.y == 0)
    // {

    //   printf("qk %f\n", qk);
    // }

    // bool is_mask = false;
    if (ti < act_time_step && tid % THREADS_PER_KEY == 0) {
      // qk_max = is_mask ? qk_max : fmaxf(qk_max, qk);
      // auto mask_bhi = params.mask_broadcast_num_heads ? bi : bhi;
      // // T mask = params.attn_mask[mask_bhi * (params.timestep + 1) + ti];
      // if (params.attn_mask) {
      //   T mask = params.attn_mask[mask_bhi * params.mask_length + ti];
      //   qk += static_cast<float>(mask);
      // }
      qk_max = fmaxf(qk_max, qk);

      qk_smem[ti] = qk;
    }
  }

#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  const int warp = tid / WARP_SIZE;
  const int lane = tid % WARP_SIZE;

  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  __syncthreads();

  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  float sum = 0.f;
  for (int ti = tid; ti < act_time_step; ti += THREADS_PER_BLOCK) {
    // bool is_mask = false;
    // float logit = is_mask ? 0.f : __expf(qk_smem[ti] - qk_max);
    float logit = __expf(qk_smem[ti] - qk_max);
    sum += logit;
    qk_smem[ti] = logit;
  }

  sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], sum);

  // FIXME(wangxi): need add 1.e-6f?
  float inv_sum = __fdividef(1.f, sum + 1.e-6f);

  for (int ti = tid; ti < act_time_step; ti += THREADS_PER_BLOCK) {
    convert_from_float(logits_smem[ti], qk_smem[ti] * inv_sum);
  }
  __syncthreads();

  // if( (threadIdx.x == 0 || threadIdx.x == 1 ) && blockIdx.x == 0 &&
  // blockIdx.y == 0)
  // {

  //   printf("softmax res %f\n", logits_smem[0]);
  // }

  constexpr int V_VEC_SIZE = Dh_MAX / THREADS_PER_VALUE;  // 128 / 16 = 8
  using V_vec = typename V_vec_<T, V_VEC_SIZE>::Type;

  // now we have got [1, seq] ，distributed in logits_smem.
  // next we compute [1, seq] * [seq, head_dim] = [1, head_dim]
  // THREADS_PER_VALUE means num of threads per value's head_dim.
  // we split the seq dimension for more cuda threads to compute.
  // vo means the first seq index processed by this cuda thread in the value.
  // vi means the head_dim index processed by this cuda thread in the value.
  // so this cuda thread compute [1, k] * [k, vi:vi+V_VEC_SIZE] and k starts
  // from vo and increases by a step V_PER_ITER.

  // THREADS_PER_VALUE == 16
  int vo = tid / THREADS_PER_VALUE;
  int vi = (tid % THREADS_PER_VALUE) * V_VEC_SIZE;

#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
  using V_vec_acum = typename V_vec_acum_fp32_<V_vec>::Type;
#else
  using V_vec_acum = V_vec;
#endif

  V_vec_acum out;
  zero(out);
  // V_PER_ITER is used to strip-mined the seq dimension.
  constexpr int V_PER_ITER =
      THREADS_PER_BLOCK / THREADS_PER_VALUE;  // 128 / 16 == 8?
  if (Dh == Dh_MAX || vi < Dh) {
    for (int ti = vo; ti < act_time_step; ti += V_PER_ITER) {
      // 8 x float16
      V_vec v;

      // update here
      v = *reinterpret_cast<const V_vec *>(
          &params.v[ti * params.num_head * Dh + vi + hi * Dh]);

#if defined(MMHA_USE_FP32_ACUM_FOR_LOGITS)
      float logit = logits_smem[ti];
      out = fma(logit, cast_to_float(v), out);
#else
      DataType_ logit = static_cast<DataType_>(logits_smem[ti]);
      // Update the partial sums.
      out = fma(logit, v, out);
#endif
    }
  }

  __syncthreads();

  // V_PER_ITER == 8
  // THREADS_PER_BLOCK = 128
  // THREADS_PER_VALUE = 16
  // V_VEC_SIZE = 8
  // if( (threadIdx.x == 0 ) && blockIdx.x == 0 && blockIdx.y == 0)
  // {

  //   printf("output %d %d %d %d\n", V_PER_ITER, THREADS_PER_BLOCK,
  //   THREADS_PER_VALUE, V_VEC_SIZE);
  // }

  // now we do the reduction in the seq dimension to get [1, head_dim].
  if (Dh == Dh_MAX || vi < Dh) {
#pragma unroll
    for (int active_groups = V_PER_ITER; active_groups >= 2;
         active_groups /= 2) {
      int midpoint = active_groups / 2;

      if (vo >= midpoint && vo < active_groups && (Dh == Dh_MAX || vi < Dh)) {
#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
        convert_from_float(
            *reinterpret_cast<V_vec *>(&out_smem[(vo - midpoint) * Dh + vi]),
            out);
#else
        *reinterpret_cast<V_vec *>(&out_smem[(vo - midpoint) * Dh + vi]) = out;
#endif
      }
      __syncthreads();
      if (vo < midpoint && (Dh == Dh_MAX || vi < Dh)) {
        out =
            add(*reinterpret_cast<const V_vec *>(&out_smem[vo * Dh + vi]), out);
      }
      __syncthreads();
    }
  }

  // write the [1, head_dim] result back to global memory.
  if (vo == 0 && (Dh == Dh_MAX || vi < Dh)) {
#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
    V_vec tmp_out;
    convert_from_float(tmp_out, out);
    store_func.template store<V_vec>(tmp_out, vi + hi * Dh);
#else

    store_func.template store<V_vec>(out, vi + hi * Dh);
#endif
  }

#else
  assert(false);
#endif
}

template <typename T>
inline size_t smem_size_in_bytes(
    const Masked_multihead_attention_params<T> &params,
    int dim_head,
    int threads_per_value,
    int threads_per_block) {
  size_t qk_sz = div_up(params.timestep + 1, 4) * 16;
  size_t logits_sz = 0;

#ifndef MMHA_USE_FP32_ACUM_FOR_LOGITS  // NOLINT
  if (sizeof(T) != 4) {
    logits_sz = div_up(params.max_seq_length, 4) * 4 * sizeof(T);
  }
#endif  // NOLINT
  size_t softmax_sz = qk_sz + logits_sz;

  int rows_per_red = threads_per_block / threads_per_value;
  size_t red_sz = rows_per_red * dim_head * sizeof(T) / 2;

  return max(softmax_sz, red_sz);
}

#define MMHA_LAUNCH_KERNEL(T,                                             \
                           Dh,                                            \
                           Dh_MAX,                                        \
                           THDS_PER_KEY,                                  \
                           THDS_PER_VALUE,                                \
                           THDS_PER_BLOCK,                                \
                           stream,                                        \
                           load_func,                                     \
                           store_func)                                    \
  size_t smem_sz =                                                        \
      smem_size_in_bytes<T>(params, Dh, THDS_PER_VALUE, THDS_PER_BLOCK);  \
  constexpr auto kernel_fn = qkv_attention_kernel<T,                      \
                                                  Dh,                     \
                                                  Dh_MAX,                 \
                                                  THDS_PER_KEY,           \
                                                  THDS_PER_VALUE,         \
                                                  THDS_PER_BLOCK,         \
                                                  decltype(load_func),    \
                                                  decltype(store_func)>;  \
  if (smem_sz > 0xc000) {                                                 \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                 \
        kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_sz); \
  }                                                                       \
  dim3 grid(params.num_head, params.batch_size);                          \
  kernel_fn<<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(                   \
      params, load_func, store_func)

template <typename T, int Dh, int Dh_MAX, typename LoadFunc, typename StoreFunc>
void q_kv_fmha_launch_kernel(const Masked_multihead_attention_params<T> &params,
                             const hipStream_t &stream,
                             LoadFunc load_func,
                             StoreFunc store_func) {
  std::cerr << "fhm launch \n";
  constexpr int THREADS_PER_VALUE = Dh_MAX * sizeof(T) / 16;

  std::cerr << "dh " << Dh << "\t" << Dh_MAX << "\t" << THREADS_PER_VALUE
            << std::endl;
  if (params.timestep < 32) {
    MMHA_LAUNCH_KERNEL(
        T, Dh, Dh_MAX, 4, THREADS_PER_VALUE, 64, stream, load_func, store_func);
  } else if (params.timestep < 2048) {
#if defined(MMHA_USE_HMMA_FOR_REDUCTION) && defined(__CUDA_ARCH__) && \
    __CUDA_ARCH__ >= 750

    std::cerr << "run here!!!\n";
    MMHA_LAUNCH_KERNEL(T,
                       Dh,
                       Dh_MAX,
                       4,
                       THREADS_PER_VALUE,
                       256,
                       stream,
                       load_func,
                       store_func);
#else
    std::cerr << "step here\n";
    MMHA_LAUNCH_KERNEL(T,
                       Dh,
                       Dh_MAX,
                       2,
                       THREADS_PER_VALUE,
                       128,
                       stream,
                       load_func,
                       store_func);
#endif
  } else {
    MMHA_LAUNCH_KERNEL(T,
                       Dh,
                       Dh_MAX,
                       1,
                       THREADS_PER_VALUE,
                       256,
                       stream,
                       load_func,
                       store_func);
  }
}

template <typename T, typename LoadFunc, typename StoreFunc>
void fmha_impl_qkv(const phi::GPUContext &dev_ctx,
                   const Masked_multihead_attention_params<T> &params,
                   int dim_head,
                   LoadFunc load_func,
                   StoreFunc store_func) {
  std::cerr << "dim head " << dim_head << std::endl;
  switch (dim_head) {
    case 16:
      q_kv_fmha_launch_kernel<T, 16, 32>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 32:
      q_kv_fmha_launch_kernel<T, 32, 32>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 64:
      q_kv_fmha_launch_kernel<T, 64, 64>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 80:
      q_kv_fmha_launch_kernel<T, 80, 128>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 96:
      q_kv_fmha_launch_kernel<T, 96, 128>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 128:
      q_kv_fmha_launch_kernel<T, 128, 128>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 192:
      q_kv_fmha_launch_kernel<T, 192, 256>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    default:
      PADDLE_THROW(
          phi::errors::Unimplemented("Dim_head = %d is unsupport!", dim_head));
  }
}

template <typename T>
void DispatchFMHA(const phi::GPUContext &dev_ctx,
                  const phi::DenseTensor &q,
                  const Masked_multihead_attention_params<T> &params,
                  int dim_head,
                  phi::DenseTensor *out_tensor) {
  std::cerr << "dispath \n";
  std::cerr << "q dtype " << q.dtype() << std::endl;
  MMHALoad<T> load_func(q.data<T>());
  MMHAStore<T> store_func(out_tensor->data<T>());
  fmha_impl_qkv(dev_ctx, params, dim_head, load_func, store_func);
}

struct NormalVersion {};
struct UnusedVersion {};

template <typename T>
struct DispatchDtypeTrait {
  using FuncVersion = NormalVersion;
};

template <>
struct DispatchDtypeTrait<int32_t> {
  using FuncVersion = UnusedVersion;
};

template <typename T, typename Context>
void QKVDispatchWithDtype(const Context &dev_ctx,
                          const DenseTensor &q,
                          const DenseTensor &k,
                          const DenseTensor &v,
                          const paddle::optional<DenseTensor> &src_mask,
                          DenseTensor *out) {
  const auto &q_dims = q.dims();
  int bsz = q_dims[0];
  int cache_bsz = q.dims()[0];
  int max_seq_len = v.dims()[1];
  int dim_head = v.dims()[3];
  int timestep = max_seq_len;
  float inv_sqrt_dh = 1. / sqrt(dim_head);

  int k_num_head = k.dims()[2];
  int v_num_head = k_num_head;
  // this num_head means query's head
  int num_head = q.dims()[2];

  std::cerr << "num head " << num_head << std::endl;
  Masked_multihead_attention_params<T> params;
  bool mask_broadcast_num_heads = true;

  dev_ctx.template Alloc<T>(out);

  params.q = q.data<T>();
  params.k = k.data<T>();
  params.v = v.data<T>();

  params.mask_broadcast_num_heads = mask_broadcast_num_heads;

  params.batch_size = bsz;
  params.cache_batch_size = cache_bsz;
  params.num_head = num_head;
  params.kv_num_head = k_num_head;
  params.timestep = timestep;

  std::cerr << "time step " << params.timestep << std::endl;
  params.inv_sqrt_dh = inv_sqrt_dh;
  std::cerr << "inv sqrt dh " << params.inv_sqrt_dh << std::endl;

  DispatchFMHA<T>(dev_ctx, q, params, dim_head, out);
}

#endif  // PADDLE_WITH_HIP

template <typename T, typename Context>
void QKVMMHAKernel(const Context &dev_ctx,
                   const DenseTensor &q,
                   const DenseTensor &k,
                   const DenseTensor &v,
                   const paddle::optional<DenseTensor> &src_mask,
                   DenseTensor *out) {
  std::cerr << "11\n";
  QKVDispatchWithDtype<T, Context>(dev_ctx, q, k, v, src_mask, out);
}

}  // namespace fusion
}  // namespace phi

#if CUDA_VERSION >= 11000
PD_REGISTER_KERNEL(qkv_mha,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::QKVMMHAKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#else
PD_REGISTER_KERNEL(masked_multihead_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::QKVMMHAKernel,
                   float,
                   phi::dtype::float16) {}
#endif
