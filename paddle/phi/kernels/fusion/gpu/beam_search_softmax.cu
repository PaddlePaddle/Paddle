#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>

#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/phi/kernels/fusion/cutlass/utils/cuda_utils.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"


namespace phi {
namespace fusion {

#define FLT_MAX 1e38
// #define DEBUG_BEAM_SEARCH_SOFTMAX

static constexpr int kBlockSizeForSmallBeamWidth = 256;
static constexpr int kMaxVocabPartForStage1FastKernel = 128;

#define CASE_K(K)                                        \
  case K:                                                \
    invokeTopKSoftMaxLauncher<T, Context, 2 * K, GROUP>( \
        dev_ctx, params, beam_group_idx, stream);        \
    break

#define DISPATCH_COMPUTE_PARTS_K(K)                      \
   case K:                                               \
    ComputeVocParts<T, 2 * K>(params);                   \
    break

template <typename T>
struct BeamSearchParams {
  // Scalar values
  int batch_size{0};
  int beam_width{0};
  int beam_group_size{0};
  int beam_group_idx{0};

  int vocab_size{0};
  int dec_stride{0};
  int max_seq_len{0};
  int end_ids_len{0};

  bool fuse_softmax{true};
  bool early_stop{false};

  int voc_parts{0};
  bool use_fast_kernel{true};
  int max_smem_per_block{0};

  T *logits{nullptr};
  const int *step_ids{nullptr};               // [BS * BM, 1]
  const int *seq_lens{nullptr};               // [BS * BM, 1]

  const int *max_dec_lens{nullptr};
  const int *end_ids{nullptr};

  const T *cum_scores{nullptr};
  const int *block_tables{nullptr};
  const int *beam_cache_ids{nullptr};         

  const float *length_penalty{nullptr};       // [BS, 1]
  const float *diversity_penalty{nullptr};    // [BS, 1]

  bool *stop_flags{nullptr};                  // [BS, 1]
  int *cache_ids_out{nullptr};                // [BS * BM, max_dec_len]
  bool *beam_finished{nullptr};               // [BS * BM, 1]
  int *block_tables_out{nullptr};             // [BS * BM, max_seq_len]
  T *cum_scores_out{nullptr};                 // [BS * BM, 1]
  int *beam_hyps_out{nullptr};                // [BS * BM, max_dec_len]
  T *beam_hyps_score_out{nullptr};            // [BS * BM, 1]

  // func out
  int *next_tokens{nullptr};
  int *parent_ids{nullptr};

  // workspace
  int *tmp_ids{nullptr};
  T *tmp_vals{nullptr};
  T *tmp_buffer{nullptr};
};

template <typename T,
          typename U,
          typename = std::enable_if_t<std::is_integral<T>::value>,
          typename = std::enable_if_t<std::is_integral<U>::value>>
auto constexpr ceilDiv(T numerator, U denominator) {
  return (numerator + denominator - 1) / denominator;
}

__device__ bool is_in_end(const int id, const int *end_ids, int length) {
  bool flag = false;
  for (int i = 0; i < length; i++) {
    if (id == end_ids[i]) {
      return true;
    }
  }
  return flag;
}

template <typename T>
__device__ __forceinline__ T apply_length_penalty(T log_prob,
                                                  int length,
                                                  float length_penalty) {
  // score = log(prob) / (length)^length_penalty.
  if (length_penalty == 0.0f || length == 1) {
    return log_prob;
  }
  return log_prob / static_cast<T>(powf(length, length_penalty));
}

// <<<batch_size, beam_group_size>>>
template <typename T, int K>
__global__ void apply_group_diversity_penalty(BeamSearchParams<T> params,
                                              const int batch_size,
                                              const int beam_width,
                                              const int beam_group_idx,
                                              const int vocab_size) {
  const int beam_group_size = K / 2;
  const int batch_idx = blockIdx.x;
  const int beam_group_sub_idx = threadIdx.x;
  const bool *beam_finished = params.beam_finished + batch_idx * beam_width;
  T *logtis = params.logits + batch_idx * beam_width * vocab_size +
              beam_group_idx * beam_group_size * vocab_size +
              beam_group_sub_idx * vocab_size;
  int *next_tokens = params.next_tokens + batch_idx * beam_width;
  // apply previous group token ids penalty
#pragma unroll
  for (int token_idx = 0; token_idx < beam_group_idx * beam_group_size;
       ++token_idx) {
    const bool finished = beam_finished[token_idx];
    if (!finished) {
      const int token_id = next_tokens[token_idx];
      logtis[token_id] -= params.diversity_penalty[batch_idx];
    }
  }
}

struct DySoftMaxStruct {
  float logit;
  float score;
};

__device__ __forceinline__ DySoftMaxStruct
reduce_softmax_op(DySoftMaxStruct a, DySoftMaxStruct b) {
  bool a_bigger = (a.logit > b.logit);
  DySoftMaxStruct bigger_m = a_bigger ? a : b;
  DySoftMaxStruct smaller_m = a_bigger ? b : a;
  DySoftMaxStruct res;
  res.score =
      bigger_m.score + smaller_m.score * expf(smaller_m.logit - bigger_m.logit);
  res.logit = bigger_m.logit;
  return res;
}

template <typename T>
struct BeamHypothesis {
  T score;
  int *seq;
  int seq_len;

  __device__ __forceinline__ void init(int *_seq,
                                       T _score,
                                       const int _max_seq_len) {
    seq = _seq;
    score = _score;
    seq_len = _max_seq_len;
  }
};

template <typename T, int K>
struct BeamHypothesesTopK {
  BeamHypothesis<T> hyps[K];
  int max_dec_len;

  __device__ __forceinline__ void init(int *_beam_hyps,
                                       T *_beam_hyps_score,
                                       const int _max_dec_len) {
    max_dec_len = _max_dec_len;
    for (int i = 0; i < K; i++) {
      // 使用默认构造函数创建默认的 BeamHypothesis 对象
      hyps[i].init(
          _beam_hyps + i * _max_dec_len, _beam_hyps_score[i], _max_dec_len);
    }
  }

  __device__ void insert(const int *token_ids,
                         int step,
                         int cur_token_id,
                         T score) {
    if (score > get_worst_score()) {
      for (int i = 0; i < step; i++) {
        hyps[K - 1].seq[i] = token_ids[i];
      }
      hyps[K - 1].seq[step] = cur_token_id;
      hyps[K - 1].score = score;

      for (int k = K - 2; k >= 0; --k) {
        if (hyps[k + 1].score > hyps[k].score) {
          T tmp_score = hyps[k].score;
          hyps[k].score = hyps[k + 1].score;
          hyps[k + 1].score = tmp_score;

          int tmp_val;
          for (int i = 0;
               i <= step && (hyps[k + 1].seq[i] > 0 || hyps[k].seq[i] > 0);
               i++) {
            tmp_val = hyps[k + 1].seq[i];
            hyps[k + 1].seq[i] = hyps[k].seq[i];
            hyps[k].seq[i] = tmp_val;
          }
        }
      }
    }
  }

  __device__ __forceinline__ T get_worst_score() { return hyps[K - 1].score; }
};

template <typename T, int K>
struct TopK {
  int ids[K];
  T vals[K];
  int parent_ids[K];

  __device__ __forceinline__ void insert(T elem, int elem_id) {
    if (elem > vals[K - 1] || (ids[K - 1] == -1) ||
        ((elem == vals[K - 1]) && (elem_id < ids[K - 1]))) {
      vals[K - 1] = elem;
      ids[K - 1] = elem_id;
    }

    for (int k = K - 2; k >= 0; --k) {
      if ((vals[k + 1] > vals[k]) || (ids[k] == -1) ||
          ((vals[k + 1] == vals[k]) && (ids[k + 1] < ids[k]))) {
        T tmp_val = vals[k];
        int tmp_id = ids[k];
        vals[k] = vals[k + 1];
        ids[k] = ids[k + 1];
        vals[k + 1] = tmp_val;
        ids[k + 1] = tmp_id;
      }
    }
  }

  __device__ __forceinline__ void insert(T elem, int elem_id, int parent_id) {
    if (elem > vals[K - 1] || (ids[K - 1] == -1) ||
        ((elem == vals[K - 1]) && (elem_id < ids[K - 1]))) {
      vals[K - 1] = elem;
      ids[K - 1] = elem_id;
      parent_ids[K - 1] = parent_id;
    }

    for (int k = K - 2; k >= 0; --k) {
      if ((vals[k + 1] > vals[k]) || (ids[k] == -1) ||
          ((vals[k + 1] == vals[k]) && (ids[k + 1] < ids[k]))) {
        T tmp_val = vals[k];
        int tmp_id = ids[k];
        int parent_id2 = parent_ids[k];
        vals[k] = vals[k + 1];
        ids[k] = ids[k + 1];
        parent_ids[k] = parent_ids[k + 1];
        vals[k + 1] = tmp_val;
        ids[k + 1] = tmp_id;
        parent_ids[k + 1] = parent_id2;
      }
    }
  }
};

template <typename T, int K>
__device__ __forceinline__ TopK<T, K> reduce_topk_op(const TopK<T, K> &a,
                                                     const TopK<T, K> &b) {
  TopK<T, K> res = a;
  for (int i = 0; i < K; ++i) res.insert(b.vals[i], b.ids[i]);
  return res;
}

template <typename T, int K>
struct TopKSoftMax {
  DySoftMaxStruct softmax_md;
  TopK<T, K> topk;
};

template <typename T, int K>
__device__ __forceinline__ TopKSoftMax<T, K> reduce_topk_softmax_op(
    const TopKSoftMax<T, K> &a, const TopKSoftMax<T, K> &b) {
  TopKSoftMax<T, K> res;
  // max_logit in block
  res.softmax_md = reduce_softmax_op(a.softmax_md, b.softmax_md);
  res.topk = reduce_topk_op(a.topk, b.topk);
  return res;
}

struct __align__(8) MD {
  float m;
  float d;
};

__device__ __forceinline__ MD reduce_md_op(MD a, MD b) {
  bool const isABigger = a.m > b.m;
  MD const bigger = isABigger ? a : b;
  MD const smaller = isABigger ? b : a;
  MD res{bigger.m, bigger.d + smaller.d * __expf(smaller.m - bigger.m)};
  return res;
}

template <typename T, int K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE, 1) __global__
    void beam_search_softmax_topk_stage1_fast(const T *logits,
                                              float *tmp_buffer,
                                              const int *end_ids,
                                              const bool *beam_finished,
                                              const int *seq_lens,
                                              int beam_width,
                                              int beam_group_idx,
                                              int vocab_size,
                                              int vocab_chunk_size) {
  constexpr int PACKED_TOP_KMD_SIZE = 2 * K + 2;
  const int beam_group_size = K / 2;
  const int tid = threadIdx.x;
  const int group_beam_batch_id = blockIdx.x;
  const int batch_id = group_beam_batch_id / beam_group_size;
  const int beam_group_sub_id = group_beam_batch_id % beam_group_size;
  const int beam_batch_id = batch_id * beam_width +
                            beam_group_idx * beam_group_size +
                            beam_group_sub_id;

  const int seq_len = seq_lens[beam_batch_id];
  const bool finished = beam_finished[beam_batch_id];

  if (seq_len < 0 || finished) {
    return;
  }

  const int section_start = vocab_chunk_size * blockIdx.y;
  const int section_end =
      std::min(section_start + vocab_chunk_size, vocab_size);
  const int valid_smem_length = section_end - section_start;
  T const MAX_T_VAL = 1e38;

  // Load element from logits to smemLogProbs, doing reduce_md and argmax
  // meanwhile Each thread is responsible for `vocab_chunk_size /
  // THREADBLOCK_SIZE` elements
  extern __shared__ char smem[];
  T *smemLogProbs = reinterpret_cast<T *>(smem);

  MD partial_md{-MAX_T_VAL, 0.0f};

  using KVPair = hipcub::KeyValuePair<int, T>;
  KVPair topKVPairPartial{vocab_size - 1, -MAX_T_VAL};
  hipcub::ArgMax argmax;

  T const *local_logits = logits + beam_batch_id * vocab_size;
#pragma unroll 1
  for (int i = section_start + tid; i < section_end; i += THREADBLOCK_SIZE) {
    T const val = local_logits[i];
    const int smem_index = i - section_start;
    smemLogProbs[smem_index] = val;
    MD new_elem_md{val, 1.0F};
    partial_md = reduce_md_op(partial_md, new_elem_md);
    KVPair new_elem_topk{smem_index, val};
    topKVPairPartial = argmax(topKVPairPartial, new_elem_topk);
  }
  __syncthreads();

  // Search the top 2K elements among `vocab_chunk_size` elements of this
  // ThreadBlock and write into smemOutput
  __shared__ float smemOutput[PACKED_TOP_KMD_SIZE];
  __shared__ int threadToUpdate;

  using BlockReduceMD = hipcub::BlockReduce<MD, THREADBLOCK_SIZE>;
  using BlockReduceTopK = hipcub::BlockReduce<KVPair, THREADBLOCK_SIZE>;

  __shared__ union {
    typename BlockReduceTopK::TempStorage topk;
    typename BlockReduceMD::TempStorage md;
  } smemReduceBuffer;

  for (int i = 0; i < 2 * beam_group_size; ++i) {
    // Pop the element with largest value to "smemOutput" per iteration
    KVPair topKVPair =
        BlockReduceTopK(smemReduceBuffer.topk).Reduce(topKVPairPartial, argmax);
    if (tid == 0) {
      // const int index = beam_batch_id * vocab_size + section_start +
      const int index = section_start + topKVPair.key;
      reinterpret_cast<int *>(smemOutput)[i] = index;
      smemOutput[K + i] = topKVPair.value;
      smemLogProbs[topKVPair.key] =
          -MAX_T_VAL;  // pollute the value of the popped element
      threadToUpdate = topKVPair.key % THREADBLOCK_SIZE;
    }
    __syncthreads();

    if (tid == threadToUpdate && i < 2 * beam_group_size - 1) {
      // The thread popped the element need to update its topKVPairPartial
      // No need to do this in the last iteration
      topKVPairPartial.key = vocab_size - 1;
      topKVPairPartial.value = -MAX_T_VAL;
      for (int index = tid; index < valid_smem_length;
           index += THREADBLOCK_SIZE) {
        topKVPairPartial =
            argmax(topKVPairPartial, {index, smemLogProbs[index]});
      }
    }
  }

  // Do reduce_md among the top 2K elements in the smemOutput and write into
  // tail of smemOutput
  auto reduce_md_func = [](const MD &a, const MD &b) {
    return reduce_md_op(a, b);
  };
  MD total_md =
      BlockReduceMD(smemReduceBuffer.md).Reduce(partial_md, reduce_md_func);
  if (tid == 0) {
    smemOutput[2 * K] = total_md.d;
    smemOutput[2 * K + 1] = total_md.m;
  }
  __syncthreads();

  // Write the smemOutput into tmp_buffer
  float *local_temp_buffer =
      tmp_buffer + group_beam_batch_id * PACKED_TOP_KMD_SIZE * gridDim.y +
      blockIdx.y * PACKED_TOP_KMD_SIZE;
#pragma unroll
  for (int i = tid; i < PACKED_TOP_KMD_SIZE; i += THREADBLOCK_SIZE) {
    local_temp_buffer[i] = smemOutput[i];
  }
}

//<<<(batch_size * beam_group_size, voc_parts), 128>>>
template <typename T, int K, int THREADBLOCK_SIZE, int PACKED_TOP_KMD_SIZE>
__global__ void beam_search_softmax_topk_stage1(BeamSearchParams<T> params,
                                                const int beam_width,
                                                const int beam_group_idx,
                                                const int vocab_size,
                                                const bool fuse_softmax) {
  const int thread_id = threadIdx.x;
  const int beam_group_size = K / 2;
  const int batch_id = blockIdx.x / beam_group_size;
  const int beam_group_sub_idx = blockIdx.x % beam_group_size;
  const int beam_batch_id = batch_id * beam_width +
                            beam_group_idx * beam_group_size +
                            beam_group_sub_idx;

  const bool finish = params.beam_finished[beam_batch_id];
  const int seq_len = params.seq_lens[beam_batch_id];
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  if (blockIdx.y == 0 && thread_id == 0) {
    printf(
        "batch %d. beam_group_sub_idx %d. beam_batch_id %d. "
        "group_beam_batch_id %d. seq_len %d. \n",
        batch_id,
        beam_group_sub_idx,
        beam_batch_id,
        blockIdx.x,
        seq_len);
  }
#endif
  // for dybatch
  if (seq_len < 0 || finish) {
    return;
  }

  // 2 * K + 2
  __shared__ float buf_s[PACKED_TOP_KMD_SIZE];

  const T MAX_T_VAL = FLT_MAX;

  const int v_local = (vocab_size + gridDim.y - 1) / gridDim.y;
  const int section_start = v_local * blockIdx.y;
  int section_end = section_start + v_local;
  section_end = (section_end > vocab_size) ? vocab_size : section_end;

  T *logits = params.logits + beam_batch_id * vocab_size;
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX

  if (blockIdx.y == 0 && thread_id == 0) {
    printf("ID %d. section_start: %d. section_end: %d. logtis:%f\n",
           blockIdx.x,
           section_start,
           section_end,
           logits[0]);
  }
#endif
  if (fuse_softmax) {
    typedef hipcub::BlockReduce<TopKSoftMax<T, K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    TopKSoftMax<T, K> partial;
    for (int i = 0; i < K; ++i) {
      partial.topk.ids[i] = -1;
      partial.topk.vals[i] = -MAX_T_VAL;
    }
    partial.softmax_md.logit = -MAX_T_VAL;
    partial.softmax_md.score = 0.0F;

// process voc_parts
#pragma unroll 1
    for (int elem_id = section_start + thread_id; elem_id < section_end;
         elem_id += THREADBLOCK_SIZE) {
      T elem = logits[elem_id];
      DySoftMaxStruct new_elem{elem, 1.0F};
      partial.softmax_md = reduce_softmax_op(partial.softmax_md, new_elem);
      partial.topk.insert(elem, elem_id);
    }
    // === old_beam_search strategy ===
    // }

    // reduce voc_parts
    TopKSoftMax<T, K> total =
        BlockReduce(temp_storage).Reduce(partial, reduce_topk_softmax_op<T, K>);

    if (thread_id == 0) {
      for (int i = 0; i < K; i++) {
        reinterpret_cast<int *>(buf_s)[i] = total.topk.ids[i];
        buf_s[K + i] = total.topk.vals[i];
      }
      buf_s[2 * K] = total.softmax_md.score;
      buf_s[2 * K + 1] = total.softmax_md.logit;
    }
  } else {
    typedef hipcub::BlockReduce<TopK<T, K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    TopK<T, K> partial;
    for (int i = 0; i < K; ++i) {
      partial.ids[i] = -1;
      partial.vals[i] = -MAX_T_VAL;
    }

#pragma unroll 1
    for (int elem_id = section_start + thread_id; elem_id < section_end;
         elem_id += THREADBLOCK_SIZE) {
      T elem = logits[elem_id];
      partial.insert(elem, elem_id);
    }

    TopK<T, K> total =
        BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, K>);

    if (thread_id == 0) {
      for (int i = 0; i < K; i++) {
        reinterpret_cast<int *>(buf_s)[i] = total.ids[i];
        buf_s[K + i] = total.vals[i];
      }
    }
  }
  __syncthreads();
  // write all the voc_parts results to tmp_buffer
  for (int elem_id = thread_id; elem_id < PACKED_TOP_KMD_SIZE;
       elem_id += THREADBLOCK_SIZE) {
    params.tmp_buffer[blockIdx.x * PACKED_TOP_KMD_SIZE * gridDim.y +
               blockIdx.y * PACKED_TOP_KMD_SIZE + elem_id] = buf_s[elem_id];
  }
}

template <typename T, int K, int THREADBLOCK_SIZE, bool IS_FAST_KERNEL>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void beam_search_softmax_topk_stage2_fast(
        int *__restrict tmp_ids,
        T *__restrict tmp_vals,
        float *__restrict tmp_buffer,
        const float *__restrict cum_scores,
        const bool *__restrict beam_finished,
        const int *__restrict seq_lens,
        const int beam_width,
        const int beam_group_idx,
        const int vocab_size,
        const int voc_parts) {
  constexpr int PACKED_TOP_KMD_SIZE = 2 * K + 2;
  constexpr int beam_group_size = K / 2;
  const int group_beam_batch_id = blockIdx.x;
  const int beam_group_sub_id = blockIdx.x % beam_group_size;
  const int batch_size = group_beam_batch_id / beam_group_size;
  const int beam_batch_id = batch_size * beam_width +
                            beam_group_idx * beam_group_size +
                            beam_group_sub_id;

  if (seq_lens[beam_batch_id] < 0 || beam_finished[beam_batch_id]) {
    return;
  }

  const int tid = threadIdx.x;
  T const MAX_T_VAL = FLT_MAX;

  using KVPair = hipcub::KeyValuePair<int, T>;
  using BlockReduceTopK = hipcub::BlockReduce<KVPair, THREADBLOCK_SIZE>;
  using BlockReduceMD = hipcub::BlockReduce<MD, THREADBLOCK_SIZE>;

  __shared__ KVPair buf_smem_kv[K];

  __shared__ union {
    typename BlockReduceTopK::TempStorage topk;
    typename BlockReduceMD::TempStorage md;
  } smemReduceBuffer;

  hipcub::ArgMax argmax;
  MD partial_md{-MAX_T_VAL, 0.0f};
  KVPair topKVPair{vocab_size - 1, -MAX_T_VAL};

  auto reduce_md_func = [](const MD &a, const MD &b) {
    return reduce_md_op(a, b);
  };

  // Load and unpack into registers through smem
  float *localTempBuffer =
      tmp_buffer + PACKED_TOP_KMD_SIZE * group_beam_batch_id * voc_parts;
  if constexpr (IS_FAST_KERNEL) {  // Use share memory instead of global memory
    extern __shared__ char smem[];
    float *smemVal = reinterpret_cast<float *>(smem);
    for (int idx = tid; idx < PACKED_TOP_KMD_SIZE * voc_parts; idx += THREADBLOCK_SIZE) {
      smemVal[idx] = localTempBuffer[idx];
    }
    localTempBuffer = smemVal;
    __syncthreads();
  }

  // Find the top 2K across all voc_parts
  for (int k = 0; k < K; ++k) {
    KVPair topKVPairPartial{vocab_size - 1, -MAX_T_VAL};
    // Only threads responsible for a chunk will do the computation
    if (tid < voc_parts) {
      for (int i = 0; i < K; ++i) {
        const int current_index = tid * PACKED_TOP_KMD_SIZE + i;
        T topValue = localTempBuffer[current_index + K];
        topKVPairPartial = argmax(topKVPairPartial, {current_index, topValue});
      }
    }

    KVPair topKVPair =
        BlockReduceTopK(smemReduceBuffer.topk).Reduce(topKVPairPartial, argmax);
    __syncthreads();

    if (tid == 0) {
      // Store kv pairs in shared mem buffer
      int temp_offset = topKVPair.key;
      int global_offset = reinterpret_cast<int *>(localTempBuffer)[temp_offset];
      topKVPair.key = global_offset;
      buf_smem_kv[k] = topKVPair;

      // Invalidate the maximum value within the chunk
      reinterpret_cast<int *>(localTempBuffer)[temp_offset] =
          vocab_size - 1;                             // id in share memory
      localTempBuffer[temp_offset + K] = -MAX_T_VAL;  // value in share memory
    }
    __syncthreads();
  }

  // Extract and reduce MD values across the chunks
  if (tid < voc_parts) {
    partial_md.d = localTempBuffer[tid * PACKED_TOP_KMD_SIZE + 2 * K];
    partial_md.m = localTempBuffer[tid * PACKED_TOP_KMD_SIZE + 2 * K + 1];
  }
  __syncthreads();

  MD total_md =
      BlockReduceMD(smemReduceBuffer.md).Reduce(partial_md, reduce_md_func);

  if (tid == 0) {
    float d_total_log = logf(total_md.d);

    for (int i = 0; i < K; ++i) {
      float val = (float)buf_smem_kv[i].value - total_md.m - d_total_log;
      tmp_ids[group_beam_batch_id * K + i] =
          buf_smem_kv[i].key;
      tmp_vals[group_beam_batch_id * K + i] =
          val + cum_scores[beam_batch_id];
    }
  }
}

#define BEAM_STAGE2_KERNEL(N_VOCAB_PART, IS_FAST_KERNEL)                     \
  do {                                                                       \
    if (IS_FAST_KERNEL && nShareMemory >= (48 << 10)) {                      \
      hipFuncSetAttribute(reinterpret_cast<const void*>(                                                  \
          beam_search_softmax_topk_stage2_fast<T),                            \
                                               K,                            \
                                               N_VOCAB_PART,                 \
                                               IS_FAST_KERNEL>,              \
          hipFuncAttributeMaxDynamicSharedMemorySize,                       \
          nShareMemory);                                                     \
    }                                                                        \
    beam_search_softmax_topk_stage2_fast<T, K, N_VOCAB_PART, IS_FAST_KERNEL> \
        <<<batch_size * beam_group_size,                                     \
           N_VOCAB_PART,                                                     \
           IS_FAST_KERNEL * nShareMemory,                                    \
           stream>>>(params.tmp_ids,                                         \
                     params.tmp_vals,                                        \
                     params.tmp_buffer,                                      \
                     params.cum_scores,                                      \
                     params.beam_finished,                                  \
                     params.seq_lens,                                        \
                     beam_width,                                             \
                     beam_group_idx,                                         \
                     vocab_size,                                             \
                     voc_parts);                                             \
  } while (0);                                                               \
  return;

template <typename T, int K>
__inline__ void beamSearchSoftmaxTopkStage2FastKernelLauncher(
    BeamSearchParams<T> &params,
    const int batch_size,
    const int beam_width,
    const int beam_group_idx,
    const int vocab_size,
    const int voc_parts,
    const int max_smem_per_block,
    hipStream_t stream) {
  constexpr int beam_group_size = K / 2;
  size_t const nShareMemory = sizeof(float) * voc_parts * (2 * K + 2) +
                              sizeof(hipcub::KeyValuePair<int, T>) * K;
  if (nShareMemory < max_smem_per_block) {  // IS_FAST_KERNEL must be a
                                            // compilation-time constant
    if (voc_parts <= 32) {
      BEAM_STAGE2_KERNEL(32, true)
    }
    if (voc_parts <= 64) {
      BEAM_STAGE2_KERNEL(64, true)
    }
    BEAM_STAGE2_KERNEL(128, true)
    // No larger branch since voc_parts <= nMaxVocabPartForStage1FastKernel
  }
  BEAM_STAGE2_KERNEL(128, false)
}

template <typename T, int K, int THREADBLOCK_SIZE>
__global__ void beam_search_softmax_topk_stage2(BeamSearchParams<T> params,
                                                const int beam_width,
                                                const int beam_group_idx,
                                                const int voc_parts,
                                                const int packed_top_kmd_size,
                                                const bool fuse_softmax) {
  const int thread_id = threadIdx.x;
  const int beam_group_size = K / 2;
  const int batch_id = blockIdx.x / beam_group_size;
  const int beam_group_sub_idx = blockIdx.x % beam_group_size;
  // int vector_id = blockIdx.x;  // batch beam index.
  const int beam_batch_id = batch_id * beam_width +
                            beam_group_idx * beam_group_size +
                            beam_group_sub_idx;
  const int group_beam_batch_id = blockIdx.x;
  // const int vector_id = blockIdx.x;
  const int PACKED_TOP_KMD_SIZE = packed_top_kmd_size;
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  // printf("--stage2: group_beam_batch_id: %d beam_batch_id: %d\n",
  // group_beam_batch_id, beam_batch_id);
#endif
  // for dybatch
  const int seq_len = params.seq_lens[beam_batch_id];
  const bool finish = params.beam_finished[beam_batch_id];

  int *tmp_ids = params.tmp_ids + group_beam_batch_id * K;
  float *tmp_vals = params.tmp_vals + group_beam_batch_id * K;
  float *tmp_buffer = params.tmp_buffer;

  const T *cum_scores = params.cum_scores + beam_batch_id;
  if (seq_len < 0 || finish) {
    return;
  }
  const T MAX_T_VAL = FLT_MAX;

  extern __shared__ char buf_s_[];
  float *buf_s = reinterpret_cast<float *>(buf_s_);
  // 当前 batch beam 的所有 voc
  tmp_buffer += group_beam_batch_id * PACKED_TOP_KMD_SIZE * voc_parts;

  if (fuse_softmax) {
    typedef hipcub::BlockReduce<TopKSoftMax<T, K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    TopKSoftMax<T, K> partial;
    for (int i = 0; i < K; ++i) {
      partial.topk.ids[i] = -1;
      partial.topk.vals[i] = -MAX_T_VAL;
    }
    partial.softmax_md.logit = -MAX_T_VAL;
    partial.softmax_md.score = 0.0F;

    for (int idx = thread_id; idx < PACKED_TOP_KMD_SIZE * voc_parts;
         idx += THREADBLOCK_SIZE) {
      buf_s[idx] = tmp_buffer[idx];
    }
    __syncthreads();

    if (threadIdx.x < voc_parts) {
      float *b_s = buf_s + thread_id * PACKED_TOP_KMD_SIZE;
      for (int i = 0; i < K; i++) {
        partial.topk.ids[i] = reinterpret_cast<int *>(b_s)[i];
        partial.topk.vals[i] = b_s[K + i];
      }
      partial.softmax_md.score = b_s[2 * K];
      partial.softmax_md.logit = b_s[2 * K + 1];
    }
    __syncthreads();

    TopKSoftMax<T, K> total =
        BlockReduce(temp_storage).Reduce(partial, reduce_topk_softmax_op<T, K>);

    if (thread_id == 0) {
      // tmp_ids += group_beam_batch_id * K;
      // tmp_vals += group_beam_batch_id * K;

      float d_total_log = logf(total.softmax_md.score);
      for (int i = 0; i < K; ++i) {
        // float val = expf((float)total.topk.vals[i] - total.softmax_md.logit -
        // d_total_log);
        float val = total.topk.vals[i] - total.softmax_md.logit - d_total_log;
        tmp_ids[i] = total.topk.ids[i];
        tmp_vals[i] = val + params.cum_scores[beam_batch_id];
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
        printf(
            "group_beam_batch_id: %d, vals: %f, logit: %f, d_total_log: %f,id: %d, val: "
            "%f, cum_log_probs: %f, res: %f\n",
            group_beam_batch_id,
            total.topk.vals[i],
            total.softmax_md.logit,
            d_total_log,
            tmp_ids[i],
            val,
            params.cum_scores[beam_batch_id],
            tmp_vals[i]);
#endif
      }
    }
  } else {
    typedef hipcub::BlockReduce<TopK<T, K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    TopK<T, K> partial;
    for (int i = 0; i < K; ++i) {
      partial.ids[i] = -1;
      partial.vals[i] = -MAX_T_VAL;
    }

    for (int idx = thread_id; idx < PACKED_TOP_KMD_SIZE * voc_parts;
         idx += THREADBLOCK_SIZE) {
      buf_s[idx] = tmp_buffer[idx];
    }
    __syncthreads();

    if (threadIdx.x < voc_parts) {
      float *b_s = buf_s + thread_id * PACKED_TOP_KMD_SIZE;
      for (int i = 0; i < K; i++) {
        partial.ids[i] = reinterpret_cast<int *>(b_s)[i];
        partial.vals[i] = b_s[K + i];
      }
    }
    __syncthreads();

    TopK<T, K> total =
        BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, K>);

    if (thread_id == 0) {
      tmp_ids += group_beam_batch_id * K;
      tmp_vals += group_beam_batch_id * K;

      for (int i = 0; i < K; ++i) {
        float val = total.vals[i];
        tmp_ids[i] = total.ids[i];
        tmp_vals[i] = val + params.cum_scores[beam_batch_id];
      }
    }
  }
}

template <typename T, int K>
void invokeBeamSearchSoftmaxTopKStage2(BeamSearchParams<T> &params,
                                       const int batch_size,
                                       const int beam_width,
                                       const int beam_group_idx,
                                       const int voc_parts,
                                       const int packed_top_kmd_size,
                                       const bool fuse_softmax,
                                       gpuStream_t stream) {
  int smem_stage2_size = voc_parts * packed_top_kmd_size * sizeof(float);
  const int beam_group_size = K / 2;
  if (voc_parts <= 32) {
    beam_search_softmax_topk_stage2<T, K, 32>
        <<<batch_size * beam_group_size, 32, smem_stage2_size, stream>>>(
            params,
            beam_width,
            beam_group_idx,
            voc_parts,
            packed_top_kmd_size,
            fuse_softmax);
    return;
  }
  if (voc_parts <= 64) {
    beam_search_softmax_topk_stage2<T, K, 64>
        <<<batch_size * beam_group_size, 64, smem_stage2_size, stream>>>(
            params,
            beam_width,
            beam_group_idx,
            voc_parts,
            packed_top_kmd_size,
            fuse_softmax);
    return;
  }
  if (voc_parts <= 128) {
    beam_search_softmax_topk_stage2<T, K, 128>
        <<<batch_size * beam_group_size, 128, smem_stage2_size, stream>>>(
            params,
            beam_width,
            beam_group_idx,
            voc_parts,
            packed_top_kmd_size,
            fuse_softmax);
    return;
  }
  if (voc_parts <= 256) {
    beam_search_softmax_topk_stage2<T, K, 256>
        <<<batch_size * beam_group_size, 256, smem_stage2_size, stream>>>(
            params,
            beam_width,
            beam_group_idx,
            voc_parts,
            packed_top_kmd_size,
            fuse_softmax);
    return;
  }
}


template <typename T, int K>
__global__ void update_beam_finished_early_stop(const T *beam_hyps_score_out,
                                                bool *beam_finished) {
  if (threadIdx.x == 0) {
    int batch_idx = blockIdx.x;

    const T *cur_beam_hyps_score = beam_hyps_score_out + batch_idx * K;
    bool *cur_beam_finished = beam_finished + batch_idx * K;
    if (cur_beam_hyps_score[K - 1] > -1e8) {
      for (int i = 0; i < K; i++) {
        cur_beam_finished[i] = true;
      }
    }
  }
}

// <<<batch_size>>>
template <typename T, int K, int THREADBLOCK_SIZE, bool GROUP>
__global__ void batch_topk(BeamSearchParams<T> params,
                           const int beam_width,
                           const int beam_group_idx,
                           const int dec_stride) {
  const bool early_stop = params.early_stop;
  const int thread_id = threadIdx.x;
  const int batch_id = blockIdx.x;
  // int block_id = blockIdx.x;  // bs
  const int beam_group_size = K / 2;
  const int beam_group_start_id =
      batch_id * beam_width + beam_group_idx * beam_group_size;

  bool *beam_finished = params.beam_finished + beam_group_start_id;
  const int *step_ids = params.step_ids + beam_group_start_id;
  int *next_tokens = params.next_tokens + beam_group_start_id;
  float *cum_scores_out = params.cum_scores_out + beam_group_start_id;
  int *parent_ids = params.parent_ids + beam_group_start_id;
  float *beam_hyps_score_out = params.beam_hyps_score_out + beam_group_start_id;

  const bool finish = beam_finished[0];
  const int step_id = step_ids[0];
  const int seq_len = params.seq_lens[beam_group_start_id];
  const int max_dec_len = params.max_dec_lens[beam_group_start_id];

  const bool last_dec_step = (step_id + 1 == max_dec_len);
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  if (finish && thread_id == 0) {
    printf("batch_topk: batch %d finish \n", beam_group_start_id);
  }
#endif

  if (thread_id == 0 && seq_len > 0 && !finish) {
    TopK<T, K> partial;
    BeamHypothesesTopK<T, K / 2> beam_hyps;

    beam_hyps.init(params.beam_hyps_out + beam_group_start_id * dec_stride,
                   params.beam_hyps_score_out + beam_group_start_id,
                   dec_stride);

    for (int i = 0; i < K; ++i) {
      partial.ids[i] = -1;
      partial.vals[i] = -FLT_MAX;
      partial.parent_ids[i] = -1;
    }
    int index = batch_id * beam_group_size * K;
    if (step_id == 0) {
      for (int i = 0; i < K; i++) {
        float score_now = apply_length_penalty(
            params.tmp_vals[index + i], step_id + 1, params.length_penalty[batch_id]);
        if (!GROUP) {
          score_now -= params.diversity_penalty[batch_id] * static_cast<float>(i + 1);
        }
        partial.insert((T)score_now, params.tmp_ids[index + i], i / K);
      }
    } else {
      for (int i = 0; i < beam_group_size * K; i++) {
        float score_now = apply_length_penalty(
            params.tmp_vals[index + i], step_id + 1, params.length_penalty[batch_id]);
        if (!GROUP) {
          score_now -= params.diversity_penalty[batch_id] * static_cast<float>(i % K + 1);
        }
        partial.insert((T)score_now, params.tmp_ids[index + i], i / K);
      }
    }
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
    for (int i = 0; i < K; ++i) {
      printf("Batch %d. TopK: %d. id:%d. val: %f. parent: %d \n",batch_id, i,
      partial.ids[i], partial.vals[i], partial.parent_ids[i]);
    }
#endif
    if (partial.vals[0] < beam_hyps.hyps[beam_group_size - 1].score) {
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
      printf("batch %d best score %f < worst_hyp_score. stop\n",
             batch_id,
             params.cum_scores[index],
             beam_hyps.hyps[beam_group_size - 1].score);
#endif
      for (int i = 0; i < beam_group_size; i++) {
        beam_finished[i] = true;
      }
      return;
    }

    int next_step_num = 0;
    for (int i = 0; i < K && next_step_num < beam_group_size; i++) {
      int parent_id = partial.parent_ids[i];
      if (is_in_end(partial.ids[i], params.end_ids, params.end_ids_len) ||
          last_dec_step) {
        if (i < beam_group_size &&
            partial.vals[i] > beam_hyps.get_worst_score()) {
          const int *beam_cache_id = params.beam_cache_ids +
                                     beam_group_start_id * dec_stride +
                                     parent_id * dec_stride;
          beam_hyps.insert(beam_cache_id,
                           step_id,
                           last_dec_step ? params.end_ids[0] : partial.ids[i],
                           partial.vals[i]);
        }

        if (early_stop && beam_hyps.get_worst_score() > -1e8) {
          // stop
          for (int i = 0; i < beam_group_size; i++) {
            beam_finished[i] = true;
          }
          return;
        }
      } else {
        next_tokens[next_step_num] = partial.ids[i];
        cum_scores_out[next_step_num] = partial.vals[i];
        parent_ids[next_step_num] = parent_id;
        next_step_num += 1;
      }
    }  // for
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
    for (int i = 0; i < K / 2; i++) {
      printf("buf: %d-%d. id:%d. val: %f. parent: %d \n",
             batch_id,
             i,
             next_tokens[i],
             cum_scores_out[i],
             parent_ids[i]);
    }
#endif
    for (int i = 0; i < beam_group_size; i++) {
      beam_hyps_score_out[i] = beam_hyps.hyps[i].score;
    }

    if (last_dec_step) {
      for (int i = 0; i < beam_group_size; i++) {
        beam_finished[i] = true;
      }
    }
  }  // if (thread_id == 0)
}

template <typename T, typename Context, int K, bool GROUP>
void invokeTopKSoftMaxLauncher(const Context &dev_ctx,
                               BeamSearchParams<T> &params,
                               int beam_group_idx,
                               gpuStream_t stream) {
  const int batch_size = params.batch_size;
  const int beam_width = params.beam_width;
  const int beam_group_size = K / 2;
  const int vocab_size = params.vocab_size;
  const bool fuse_softmax = params.fuse_softmax;
  const int voc_parts = params.voc_parts;
  constexpr int dev_id = 0;

  // only in group_beam_search
  if (beam_width > beam_group_size && beam_group_idx != 0) {
    apply_group_diversity_penalty<T, K>
        <<<batch_size, beam_group_size, 0, stream>>>(
            params, batch_size, beam_width, beam_group_idx, vocab_size);
  }

  // == Step1 == : stage1
  if (params.use_fast_kernel) {
    constexpr int block_size =
      (K < 16) ? ((K < 8) ? kBlockSizeForSmallBeamWidth : 128) : 64;
    const int vocab_chunk_size = (vocab_size + voc_parts - 1) / voc_parts;
    const int dyn_smem_size = sizeof(T) * vocab_chunk_size;
    VLOG(1) << "Stage1 kernel dyn_smem_size: " << dyn_smem_size;
    if (dyn_smem_size >= (48 << 10)) {
      hipFuncSetAttribute(reinterpret_cast<const void*>(beam_search_softmax_topk_stage1_fast<T), K, block_size>,
                            hipFuncAttributeMaxDynamicSharedMemorySize,
                            dyn_smem_size);
    }
    VLOG(1) << "voc_parts: " << voc_parts;

    dim3 grid(batch_size * beam_group_size, voc_parts);
    beam_search_softmax_topk_stage1_fast<T, K, block_size>
        <<<grid, block_size, dyn_smem_size, stream>>>(params.logits,
                                                      params.tmp_buffer,
                                                      params.end_ids,
                                                      params.beam_finished,
                                                      params.seq_lens,
                                                      beam_width,
                                                      beam_group_idx,
                                                      vocab_size,
                                                      vocab_chunk_size);
  } else {
    constexpr int block_size = 128;
    VLOG(1) << "Old Stage1 kernel";
    dim3 grid(batch_size * beam_group_size, voc_parts);
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        beam_search_softmax_topk_stage1<float), K, block_size, 2 * K + 2>,
        hipFuncAttributePreferredSharedMemoryCarveout,
        cudaSharedmemCarveoutMaxL1);
    if (fuse_softmax) {
#ifdef PADDLE_WITH_CUDA
      hipFuncSetAttribute(reinterpret_cast<const void*>(
          beam_search_softmax_topk_stage1<T), K, block_size, 2 * K + 2>,
          hipFuncAttributePreferredSharedMemoryCarveout,
          cudaSharedmemCarveoutMaxL1);
#else
      // cudaSharedmemCarveoutMaxL1 equal to 0
      hipFuncSetAttribute(reinterpret_cast<const void*>(
          reinterpret_cast<void *>(
              beam_search_softmax_topk_stage1<T), K, block_size, 2 * K + 2>),
          hipFuncAttributePreferredSharedMemoryCarveout,
          0);
#endif
      // （bs, bm, voc_parts, 2 * K + 2）
      beam_search_softmax_topk_stage1<T, K, block_size, 2 * K + 2>
          <<<grid, block_size, 0, stream>>>(params,
                                            beam_width,
                                            beam_group_idx,
                                            vocab_size,
                                            fuse_softmax);
    } else {
#ifdef PADDLE_WITH_CUDA
      hipFuncSetAttribute(reinterpret_cast<const void*>(
          beam_search_softmax_topk_stage1<T), K, block_size, 2 * K>,
          hipFuncAttributePreferredSharedMemoryCarveout,
          cudaSharedmemCarveoutMaxL1);
#else
      // cudaSharedmemCarveoutMaxL1 equal to 0
      hipFuncSetAttribute(reinterpret_cast<const void*>(
          reinterpret_cast<void *>(
              beam_search_softmax_topk_stage1<T), K, block_size, 2 * K>),
          hipFuncAttributePreferredSharedMemoryCarveout,
          0);
#endif
      // （bs, bm, voc_parts, 2 * K）
      beam_search_softmax_topk_stage1<T, K, block_size, 2 * K>
          <<<grid, block_size, 0, stream>>>(params,
                                            beam_width,
                                            beam_group_idx,
                                            vocab_size,
                                            fuse_softmax);
    }
  }

  // Reserved for debug
  // invokeBeamSearchSoftmaxTopKStage2<T, K>(params,
  //                                         batch_size,
  //                                         beam_width,
  //                                         beam_group_idx,
  //                                         voc_parts,
  //                                         packed_top_kmd_size,
  //                                         fuse_softmax,
  //                                         stream);
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
  printf("======== num %dth for loop before stage2 =======\n", beam_group_idx);
  int total_ele = batch_size * beam_width;  // hard code here
  int parent_ids_arr[total_ele];
  hipMemcpy(parent_ids_arr,
             params.parent_ids,
             total_ele * sizeof(int),
             hipMemcpyDeviceToHost);
  printf("parent_ids_arr total: \n");
  for (int i = 0; i < total_ele; i++) {
    printf("%d-%f. ", parent_ids_arr[i], (float*)reinterpret_cast<float*>(parent_ids_arr+i));
    if ((i + 1) % 10 == 0) {
      printf("\n");
    }
  }
  printf("\n");
  int packed_top_kmd_size = 2 * K;
  if (fuse_softmax) {
    packed_top_kmd_size += 2;
  }
  const int tmp_buffer_size =
      batch_size * beam_group_size * voc_parts * packed_top_kmd_size;

  VLOG(0) << "tmp_buffer_size invoke: " << tmp_buffer_size;
  float* tmp_buffer_cpu = (float*)malloc(tmp_buffer_size * sizeof(float));
  printf("point3 %p\n", params.tmp_buffer);
  hipMemcpy(tmp_buffer_cpu, params.tmp_buffer, tmp_buffer_size * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < batch_size; i++) {
    printf("--- batch %d ---\n", i);
    for (int j = 0; j < beam_group_size; j++) {
      printf("  -- sub_group_id %d --\n", j);
      for (int k = 0; k < voc_parts; k++) {
        printf("    - voc part id - %d - \n    ", k);
        for (int elem_id = 0; elem_id < K; elem_id++) {
          printf("%dth idx:%d.  ", elem_id, *reinterpret_cast<int*>(tmp_buffer_cpu + i * beam_group_size * voc_parts * packed_top_kmd_size + 
                  j * voc_parts * packed_top_kmd_size + k * packed_top_kmd_size + elem_id));
        }
        printf("\n    ");
        for (int elem_id = K ; elem_id < packed_top_kmd_size; elem_id++) {
          printf("%dth val:%f.  ", elem_id, tmp_buffer_cpu[i * beam_group_size * voc_parts * packed_top_kmd_size + 
                  j * voc_parts * packed_top_kmd_size + k * packed_top_kmd_size + elem_id]);

        }
        printf("\n");
      }
    }
  }
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif
  beamSearchSoftmaxTopkStage2FastKernelLauncher<T, K>(params,
                                                      batch_size,
                                                      beam_width,
                                                      beam_group_idx,
                                                      vocab_size,
                                                      voc_parts,
                                                      params.max_smem_per_block,
                                                      stream);

  batch_topk<T, K, 32, GROUP><<<batch_size, 32, 0, stream>>>(
      params, beam_width, beam_group_idx, params.dec_stride);
  // === old_beam_search strategy ===
  // }

#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
  printf("======== num %dth for loop =======\n", beam_group_idx);
  // int total_ele = batch_size * beam_width;  // hard code here
  // int parent_ids_arr[total_ele];
  hipMemcpy(parent_ids_arr,
             params.parent_ids,
             total_ele * sizeof(int),
             hipMemcpyDeviceToHost);
  printf("parent_ids_arr total: \n");
  for (int i = 0; i < total_ele; i++) {
    printf("%d. ", parent_ids_arr[i]);
    if ((i + 1) % 10 == 0) {
      printf("\n");
    }
  }
  printf("\n");
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());

  bool beam_finished_arr[total_ele];
  hipMemcpy(beam_finished_arr,
             params.beam_finished,
             sizeof(bool) * total_ele,
             hipMemcpyDeviceToHost);
  printf("beam_finished total: \n");
  for (int i = 0; i < total_ele; i++) {
    printf("%d. ", beam_finished_arr[i]);
    if ((i + 1) % 10 == 0) {
      printf("\n");
    }
  }
  printf("\n");
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());

  int next_tokens_arr[total_ele];
  hipMemcpy(next_tokens_arr,
             params.next_tokens,
             sizeof(int) * total_ele,
             hipMemcpyDeviceToHost);
  printf("next_tokens total: \n");
  for (int i = 0; i < total_ele; i++) {
    printf("%d. ", next_tokens_arr[i]);
    if ((i + 1) % 10 == 0) {
      printf("\n");
    }
  }
  printf("\n");
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif
}

template <typename T, typename Context, bool GROUP>
void invokeTopkSoftMax(const Context &dev_ctx,
                       BeamSearchParams<T> &params,
                       int beam_group_idx,
                       gpuStream_t stream) {
  switch (params.beam_group_size) {
    CASE_K(1);
    CASE_K(2);
    CASE_K(3);
    CASE_K(4);
    CASE_K(5);
    CASE_K(6);
    CASE_K(7);
    CASE_K(8);
    CASE_K(9);
    CASE_K(10);
    CASE_K(11);
    CASE_K(12);
    CASE_K(13);
    CASE_K(14);
    CASE_K(15);
    CASE_K(16);
    default:
      PADDLE_THROW(errors::InvalidArgument(
            "Beam_group_size/Beam_width must <= 16, but get %d",
            params.beam_group_size));
  }
}


template<typename T, int K>
void ComputeVocParts(BeamSearchParams<T> &params) {
  int dev_id = 0;
  const int block_size =
      (K < 16) ? ((K < 8) ? kBlockSizeForSmallBeamWidth : 128) : 64;
  int max_active_blocks = -1;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &max_active_blocks,
      beam_search_softmax_topk_stage1_fast<float, K, block_size>,
      block_size,
      0);

  int max_smem_per_sm = -1;
  int max_smem_per_block = -1;
  hipDeviceGetAttribute(
      &max_smem_per_sm, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev_id);
  hipDeviceGetAttribute(
      &max_smem_per_block, hipDeviceAttributeSharedMemPerBlockOptin, dev_id);
  hipFuncAttributes attr;
  hipFuncGetAttributes(
      &attr, reinterpret_cast<const void*>(beam_search_softmax_topk_stage1_fast<float), K, block_size>);
  const int static_smem = attr.sharedSizeBytes;
  const int max_dyn_smem_per_block = max_smem_per_block - static_smem;

  if (sizeof(T) * params.vocab_size >
      max_dyn_smem_per_block * kMaxVocabPartForStage1FastKernel) {
    VLOG(1) << "Vocab size is too large. Back to old kernel.";
  }

  VLOG(1) << "max_active_blocks: " << max_active_blocks;
  const int driver_smem_per_block = max_smem_per_sm - max_smem_per_block;
  const int extra_smem = driver_smem_per_block + static_smem;
  VLOG(1) << "max_smem_per_sm: " << max_smem_per_sm
          << ". max_smem_per_block: " << max_smem_per_block
          << ". extra_smem: " << extra_smem;
  int voc_parts = kMaxVocabPartForStage1FastKernel + 1;
  VLOG(1) << "Start compute voc_parts";
  for (int n_block = max_active_blocks - 1; n_block > 0 && voc_parts > kMaxVocabPartForStage1FastKernel; --n_block) {
    int dyn_smem_size = max_smem_per_sm / n_block - extra_smem;
    dyn_smem_size -= dyn_smem_size % sizeof(T);
    voc_parts = ceilDiv(sizeof(T) * params.vocab_size, dyn_smem_size);
    VLOG(1) << "n_block: " << n_block << ". dyn_smem_size:" << dyn_smem_size
            << ". voc_parts: " << voc_parts;
  }

  if (!params.fuse_softmax || voc_parts > kMaxVocabPartForStage1FastKernel) {
    params.use_fast_kernel = false;
    VLOG(1) << "Vocab size is too big for shared-memory. Falling back to the old algorithm";
    int sm_count;
    hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev_id);
    const int max_act_blocks_per_sm = 4;
    const int max_act_blocks_per_wave = sm_count * max_act_blocks_per_sm;
    const int gridx = params.batch_size * K / 2;
    const int max_part_num = (max_act_blocks_per_wave + gridx - 1) / gridx;
    voc_parts = min(128, max_part_num);
  }
  params.voc_parts = voc_parts;
  params.max_smem_per_block = max_smem_per_block;
  VLOG(1) << "BeamSearch Pre-compute. voc_parts: " << params.voc_parts << ". use_fast_kernel: " << params.use_fast_kernel;
}

template<typename T>
void DispatchComputeVocParts(BeamSearchParams<T> &params){
  switch (params.beam_group_size) {
    DISPATCH_COMPUTE_PARTS_K(1);
    DISPATCH_COMPUTE_PARTS_K(2);
    DISPATCH_COMPUTE_PARTS_K(3);
    DISPATCH_COMPUTE_PARTS_K(4);
    DISPATCH_COMPUTE_PARTS_K(5);
    DISPATCH_COMPUTE_PARTS_K(6);
    DISPATCH_COMPUTE_PARTS_K(7);
    DISPATCH_COMPUTE_PARTS_K(8);
    DISPATCH_COMPUTE_PARTS_K(9);
    DISPATCH_COMPUTE_PARTS_K(10);
    DISPATCH_COMPUTE_PARTS_K(11);
    DISPATCH_COMPUTE_PARTS_K(12);
    DISPATCH_COMPUTE_PARTS_K(13);
    DISPATCH_COMPUTE_PARTS_K(14);
    DISPATCH_COMPUTE_PARTS_K(15);
    DISPATCH_COMPUTE_PARTS_K(16);
    default:
      PADDLE_THROW(errors::InvalidArgument(
            "Beam_group_size/Beam_width must <= 16, but get %d",
            params.beam_group_size));
  }
}

template <typename T>
__global__ void update_beam_search_params_kernel(BeamSearchParams<T> params) {
  int bb_id = blockIdx.y;
  int time_step = threadIdx.x + blockIdx.x * blockDim.x;

  const bool finished = params.beam_finished[bb_id];
  const int seq_len = params.seq_lens[bb_id];

  if (bb_id >= params.beam_width * params.batch_size) {
    return;
  }

  if (finished || seq_len < 0) {
    return;
  }

  const int beam_group_size = params.beam_group_size;
  const int max_seq_len = params.max_seq_len;
  const int dec_stride = params.dec_stride;

  const int batch_group_id = bb_id / beam_group_size;

  const int max_dec_len = params.max_dec_lens[bb_id];
  const int src_beam = params.parent_ids[bb_id];
  const int step = params.step_ids[bb_id];

  const int *block_tables = params.block_tables;
  int *block_tables_out = params.block_tables_out;
  const int *cache_ids = params.beam_cache_ids;
  int *cache_ids_out = params.cache_ids_out;
  const int *next_tokens = params.next_tokens;

  const int beam_group_sub_id = bb_id % beam_group_size;
  // const int src_bb_id = batch_group_id * beam_group_size + src_beam;

  if (time_step < min(max_seq_len, seq_len + 1)) {
    const uint block_tables_tgt_offset =
        batch_group_id * beam_group_size * max_seq_len +
        beam_group_sub_id * max_seq_len + time_step;
    const uint block_tables_src_offset =
        batch_group_id * beam_group_size * max_seq_len +
        src_beam * max_seq_len + time_step;
    block_tables_out[block_tables_tgt_offset] =
        block_tables[block_tables_src_offset];
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  printf("block_table. src_beam %d. time_step. %d. bid:%d, subID:%d, tgt_offset:%d, src_offset:%d, val: %d \n", src_beam, time_step, bb_id / params.beam_width, beam_group_sub_id, block_tables_tgt_offset, block_tables_src_offset, block_tables[block_tables_src_offset]);
#endif
    if (time_step < min(step + 1, max_dec_len)) {
      const uint cache_ids_tgt_offset =
          batch_group_id * beam_group_size * dec_stride +
          beam_group_sub_id * dec_stride + time_step;
      const uint cache_ids_src_offset =
          batch_group_id * beam_group_size * dec_stride +
          src_beam * dec_stride + time_step;
      cache_ids_out[cache_ids_tgt_offset] =
          (time_step == step) ? next_tokens[bb_id]
                              : cache_ids[cache_ids_src_offset];
#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  printf("cache_ids. src_beam %d. time_step. %d, bid:%d, subID:%d, tgt_offset:%d, src_offset:%d, val:%d\n", src_beam, time_step, bb_id / params.beam_width, beam_group_sub_id, cache_ids_tgt_offset, cache_ids_src_offset, cache_ids_out[cache_ids_tgt_offset]);
#endif
    }
  }
}

template <typename T>
__global__ void update_stop_flags(BeamSearchParams<T> params) {
  int bid = blockIdx.x;
  const int beam_width = params.beam_width;
  const int beam_group_size = params.beam_group_size;
  const bool* beam_finished = params.beam_finished + beam_width * bid;
  bool* stop_flags = params.stop_flags + beam_width * bid;
  bool finished = true;
  if (threadIdx.x == 0 && !stop_flags[0]) {
#pragma unroll
    for (int i = 0; i < beam_width; i += beam_group_size) {
      finished &= beam_finished[i];
    }
    if (finished) {
#pragma unroll
      for (int i = 0; i < beam_width; i++) {
        stop_flags[i] = true;
      }
    }
  }
}

template <typename T>
void updateBeamSearchParams(BeamSearchParams<T> &params, hipStream_t stream) {
  const dim3 block(32);
  const dim3 grid((params.max_seq_len + block.x - 1) / block.x,
                  params.batch_size * params.beam_width);

  update_beam_search_params_kernel<<<grid, block, 0, stream>>>(params);

  const dim3 grid_2(params.batch_size);
  update_stop_flags<<<grid_2, 1, 0, stream>>>(params);
}

/*****
liuzichang01(Note): In order to adapt to the model structure of 5.2 without
adding while op and without affecting the speed. Use a 'fake inplace' method
here. Not elegant but useful ︸_︸.
*****/
template <typename T, typename Context>
void BeamSearchSoftmaxKernel(const Context &dev_ctx,
                             const DenseTensor &logits,
                             const DenseTensor &seq_lens,  
                             const DenseTensor &stop_flags,       // inplace
                             const DenseTensor &end_ids,
                             const DenseTensor &step_ids,  
                             const DenseTensor &max_dec_lens,
                             const DenseTensor &block_tables,     // inplace
                             const DenseTensor &cum_scores,       // inplace
                             const DenseTensor &beam_cache_ids,   // inplace
                             const DenseTensor &beam_hyps,        // inplace
                             const DenseTensor &beam_hyps_score,  // inplace
                             const DenseTensor &beam_finished,    // inplace
                             const DenseTensor &beam_width,
                             const DenseTensor &beam_group_num,
                             const DenseTensor &length_penalty,
                             const DenseTensor &diversity_penalty,
                             bool fuse_softmax,
                             bool early_stop,
                             DenseTensor *next_tokens,
                             DenseTensor *parent_ids) {
  // PADDLE_ENFORCE_EQ(beam_width % beam_group_num, 
  //                   0, 
  //                   platform::errors::InvalidArgument(
  //                     "beam_width must be divisible by beam_group_num."
  //                   ));
  
  const auto &logits_dims = logits.dims();

  int beam_width_scalar;
  hipMemcpyAsync(&beam_width_scalar, beam_width.data<int>(), sizeof(int), hipMemcpyDeviceToHost, dev_ctx.stream());

  int beam_group_num_scalar;
  hipMemcpyAsync(&beam_group_num_scalar, beam_group_num.data<int>(), sizeof(int), hipMemcpyDeviceToHost, dev_ctx.stream());

  int beam_batch_size = logits_dims[0];
  int batch_size = beam_batch_size / beam_width_scalar;
  int vocab_size = logits_dims[1];
  const int max_seq_len = block_tables.dims()[1];
  // liuzichang: In some cases, the length of Tensor is longer than max_dec_lens
  const int dec_stride = beam_hyps.dims()[1];
  const int end_ids_len = end_ids.dims()[0];
  const int beam_group_size = beam_width_scalar / beam_group_num_scalar;

#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
  VLOG(2) << "beam_width: " << beam_width_scalar << ", beam_group_num: " << beam_group_num_scalar;
  VLOG(2) << "bsf-input -- logits: " << logits;
  VLOG(2) << "bsf-input -- cum_scores: " << cum_scores;
  VLOG(2) << "bsf-input -- seq_lens: " << seq_lens;
  VLOG(2) << "bsf-input -- beam_finished: " << beam_finished;
  VLOG(2) << "bsf-input -- end_ids: " << end_ids;
  VLOG(2) << "bsf-input -- step_ids: " << step_ids;
  VLOG(2) << "bsf-input -- beam_cache_ids: " << beam_cache_ids;
  VLOG(2) << "bsf-input -- block_tables: " << block_tables;
  VLOG(2) << "bsf-input -- beam_hyps: " << beam_hyps;
  VLOG(2) << "bsf-input -- beam_hyps_score: " << beam_hyps_score;
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif

  dev_ctx.template Alloc<int>(next_tokens);
  dev_ctx.template Alloc<int>(parent_ids);
  hipMemset(parent_ids->data<int>(), 0, beam_batch_size * sizeof(int));

  DenseTensor cum_scores_ori;
  cum_scores_ori.Resize(cum_scores.dims());
  dev_ctx.template Alloc<T>(&cum_scores_ori);

  DenseTensor beam_cache_ids_ori;
  beam_cache_ids_ori.Resize(beam_cache_ids.dims());
  dev_ctx.template Alloc<int>(&beam_cache_ids_ori);

  DenseTensor block_tables_ori;
  block_tables_ori.Resize(block_tables.dims());
  dev_ctx.template Alloc<int>(&block_tables_ori);

  phi::Copy(
      dev_ctx, beam_cache_ids, dev_ctx.GetPlace(), false, &beam_cache_ids_ori);
  phi::Copy(dev_ctx, cum_scores, dev_ctx.GetPlace(), false, &cum_scores_ori);
  phi::Copy(
      dev_ctx, block_tables, dev_ctx.GetPlace(), false, &block_tables_ori);

  const int tmp_size = batch_size * beam_group_size * beam_group_size * 2;
  DenseTensor tmp_topk_id;
  tmp_topk_id.Resize(phi::make_ddim({tmp_size}));
  dev_ctx.template Alloc<int>(&tmp_topk_id);

  DenseTensor tmp_topk_val;
  tmp_topk_val.Resize(phi::make_ddim({tmp_size}));
  dev_ctx.template Alloc<T>(&tmp_topk_val);

  BeamSearchParams<T> params;
  params.batch_size = batch_size;
  params.beam_width = beam_width_scalar;
  params.beam_group_size = beam_group_size;

  params.vocab_size = vocab_size;
  params.dec_stride = dec_stride;
  params.max_seq_len = max_seq_len;
  params.end_ids_len = end_ids_len;

  params.fuse_softmax = fuse_softmax;
  params.early_stop = early_stop;

  // Only Read
  params.step_ids = step_ids.data<int>();
  params.seq_lens = seq_lens.data<int>();
  params.max_dec_lens = max_dec_lens.data<int>();
  params.end_ids = end_ids.data<int>();
  params.length_penalty = length_penalty.data<float>();
  params.diversity_penalty = diversity_penalty.data<float>();

  params.cum_scores = cum_scores_ori.data<T>();
  params.block_tables = block_tables_ori.data<int>();
  params.beam_cache_ids = beam_cache_ids_ori.data<int>();

  // Write
  params.logits = const_cast<T *>(logits.data<T>());
  params.cache_ids_out = const_cast<int *>(beam_cache_ids.data<int>());
  params.block_tables_out = const_cast<int *>(block_tables.data<int>());
  params.cum_scores_out = const_cast<T *>(cum_scores.data<T>());
  params.beam_hyps_out = const_cast<int *>(beam_hyps.data<int>());
  params.beam_hyps_score_out = const_cast<T *>(beam_hyps_score.data<T>());
  params.beam_finished = const_cast<bool *>(beam_finished.data<bool>());
  params.stop_flags = const_cast<bool *>(stop_flags.data<bool>());

  params.next_tokens = next_tokens->data<int>();
  params.parent_ids = parent_ids->data<int>();

  params.tmp_ids = tmp_topk_id.data<int>();
  params.tmp_vals = tmp_topk_val.data<T>();

  DispatchComputeVocParts<T>(params);
  // allocate workspace 
  const int tmp_id_val_size = batch_size * beam_group_size * beam_group_size * 2;
  const int packed_top_kmd_size = fuse_softmax ? 2 * 2 * beam_group_size + 2 : 2 * 2 * beam_group_size;
  const int tmp_stage1_to_stage2_size =
      batch_size * beam_group_size * params.voc_parts * packed_top_kmd_size;

  const int workspace_size = tmp_id_val_size * 2 + tmp_stage1_to_stage2_size;
  DenseTensor wsp_buffer_tensor;
  wsp_buffer_tensor.Resize(phi::make_ddim({workspace_size}));
  dev_ctx.template Alloc<float>(&wsp_buffer_tensor);
  params.tmp_ids = reinterpret_cast<int*>(wsp_buffer_tensor.data<float>());
  params.tmp_vals = wsp_buffer_tensor.data<float>() + tmp_id_val_size;
  params.tmp_buffer = wsp_buffer_tensor.data<float>() + 2 * tmp_id_val_size;
  VLOG(2) << "tmp_id_val_size: " << tmp_id_val_size << ". tmp_stage1_to_stage2_size: " << tmp_stage1_to_stage2_size;

  for (int beam_group_idx = 0; beam_group_idx < beam_group_num_scalar; ++beam_group_idx) {
    if (beam_group_num_scalar == 1) {
      invokeTopkSoftMax<T, Context, false>(
          dev_ctx, params, beam_group_idx, dev_ctx.stream());
    } else {
      invokeTopkSoftMax<T, Context, true>(
          dev_ctx, params, beam_group_idx, dev_ctx.stream());
    }
  }
  updateBeamSearchParams<T>(params, dev_ctx.stream());

#ifdef DEBUG_BEAM_SEARCH_SOFTMAX
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
  VLOG(2) << "bsf -- next_token: " << *next_tokens;
  VLOG(2) << "bsf -- beam_finished: " << beam_finished;
  VLOG(2) << "bsf -- parent_ids: " << *parent_ids;
  VLOG(2) << "bsf -- seq_lens_out: " << seq_lens;
  VLOG(2) << "bsf -- step_ids_out: " << step_ids;
  VLOG(2) << "bsf -- cache_ids_out: " << beam_cache_ids;
  VLOG(2) << "bsf -- block_tables_out: " << block_tables;
  VLOG(2) << "bsf -- beam_hyps_out: " << beam_hyps;
  VLOG(2) << "bsf -- beam_hyps_score_out: " << beam_hyps_score;
  PADDLE_ENFORCE_GPU_SUCCESS(hipDeviceSynchronize());
#endif
}

}  // namespace fusion
}  // namespace phi

PD_REGISTER_KERNEL(beam_search_softmax,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::BeamSearchSoftmaxKernel,
                   float) {}  // only supports float
