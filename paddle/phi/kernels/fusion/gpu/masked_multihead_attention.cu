#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/aligned_vector.h"
#include "paddle/phi/kernels/fusion/gpu/mmha_util.cu.h"

namespace phi {
namespace fusion {

#ifndef PADDLE_WITH_HIP

#define MMHA_USE_FP32_ACUM_FOR_LOGITS
#define MMHA_USE_FP32_ACUM_FOR_OUT
#define MMHA_USE_FP32_ACUM_FOR_FMA

template <typename T>
__device__ __inline__ T ClipFunc(const T v, const T min, const T max) {
  if (v > max) return max;
  if (v < min) return min;
  return v;
}

constexpr unsigned int str2int(const char *str, int h = 0) {
  return !str[h] ? 5381 : (str2int(str, h + 1) * 33) ^ str[h];
}

template <typename InType, typename OutType>
__forceinline__ __device__ OutType QuantHelperFunc(const InType input,
                                                   const float scale,
                                                   const int round_type,
                                                   const float max_bound,
                                                   const float min_bound) {
  float quant_value = max_bound * scale * input;

  if (round_type == 0) {
    quant_value = static_cast<float>(rint(quant_value));
  } else {
    quant_value = static_cast<float>(round(quant_value));
  }
  return static_cast<OutType>(
      ClipFunc<float>(quant_value, min_bound, max_bound));
}

template <typename T>
struct Masked_multihead_attention_params {
  // output buffer, [B, 1(seq_len), num_head * dim_head]
  T *out;
  // qkv_out, [B, 1(seq_len), 3, num_head * dim_head]
  const T *qkv;
  // bias, [3, num_head, dim_head]
  T *qkv_bias;
  // [bsz, seq_len]
  const int *cum_offsets;
  // TODO(wangxi): optimize with input_lengths and max_input_len?
  // [bsz, 1, 1, time_step(cache_seq_length)+1]
  const T *attn_mask;
  int mask_length;
  // whether to broadcast num_heads(2nd) dimension for attn_mask
  // in MMHA, if false, attn_mask shape should be
  // [bsz, num_heads, 1, time_step(cache_seq_length)+1]
  bool mask_broadcast_num_heads;

  // [2, B, num_head, max_seq_len(valid cache_seq_len), dim_head]
  // k [B, num_head, dim_head/x, max_seq_len, x], that is `seq_len` first
  // v [B, num_head, max_seq_len, dim_head]
  T *cache_kv;
  // [B, max_seq_len]
  const int *beam_cache_offset = nullptr;

  const int *sequence_lengths{nullptr};

  // The RoPE embedding, [2, B, rotary_seq_len, 1, dim_head]
  // rotary_emb_dims = 1 if pos_ids_extra is null else 2
  const float *rotary_emb;
  int rotary_emb_dims;
  int rotary_seq_len = 1;

  int batch_size;  // batch * beam
  int beam_width;
  int cache_batch_size;
  int num_head;
  // k_num_head and v_num_head must be equal, we unify them.
  // kv_num_head = k_num_head && kv_num_head == v_num_head
  int kv_num_head;
  int timestep;  // cache_seq_length
  int seq_len;
  int max_seq_length;

  // 1.f / sqrt(Dh)
  float inv_sqrt_dh;

  bool add_qkv_bias;
  bool neox_rotary_style;
};

#ifdef MMHA_USE_FP32_ACUM_FOR_FMA
template <typename T>
struct K_vec_acum_fp32_ {};

template <>
struct K_vec_acum_fp32_<uint32_t> {
  using Type = float2;
};
#endif

#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
template <typename T>
struct V_vec_acum_fp32_ {};
// template <> struct V_vec_acum_fp32_<float>  { using Type = float;  };
// template <> struct V_vec_acum_fp32_<float2> { using Type = float2; };
template <>
struct V_vec_acum_fp32_<float4> {
  using Type = float4;
};
// template <> struct V_vec_acum_fp32_<uint32_t> { using Type = float2;   };
// template <> struct V_vec_acum_fp32_<uint2   > { using Type = Float4_;  };
template <>
struct V_vec_acum_fp32_<uint4> {
  using Type = Float8_;
};

#ifdef ENABLE_BF16
template <>
struct V_vec_acum_fp32_<__hip_bfloat162> {
  using Type = float2;
};
template <>
struct V_vec_acum_fp32_<bf16_4_t> {
  using Type = Float4_;
};
template <>
struct V_vec_acum_fp32_<bf16_8_t> {
  using Type = Float8_;
};
#endif  // ENABLE_BF16

#endif

// clang-format on

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int THREADS_PER_KEY, typename K_vec, int N>
inline __device__ float qk_dot_(const K_vec (&q)[N],
                                const K_vec (&k)[N],
                                float inv_sqrt_dh) {
  K_vec inv_q = mul<K_vec, K_vec, float>(q[0], inv_sqrt_dh);
  K_vec qk_vec = mul<K_vec, K_vec, K_vec>(inv_q, k[0]);
#pragma unroll
  for (int ii = 1; ii < N; ++ii) {
    inv_q = mul<K_vec, K_vec, float>(q[ii], inv_sqrt_dh);
    qk_vec = fma(inv_q, k[ii], qk_vec);
  }

  float qk = sum(qk_vec);
#pragma unroll
  for (int mask = THREADS_PER_KEY / 2; mask >= 1; mask /= 2) {
    qk += __shfl_xor_sync(uint32_t(-1), qk, mask);
  }
  return qk;
}

inline __device__ float4 hmma_fp32_tensorcore(const uint2 &a, uint32_t b) {
  float4 c;
  float zero = 0.f;
  asm volatile(
      "mma.sync.aligned.m16n8k8.row.col.f32.f16.f16.f32 \n"
      "    {%0, %1, %2, %3}, \n"
      "    {%4, %5}, \n"
      "    {%6}, \n"
      "    {%7, %7, %7, %7}; \n"

      : "=f"(c.x), "=f"(c.y), "=f"(c.z), "=f"(c.w)
      : "r"(a.x) "r"(a.y), "r"(b), "f"(zero));
  return c;
}

template <int N>
inline __device__ float qk_hmma_dot_(const uint32_t (&q)[N],
                                     const uint32_t (&k)[N],
                                     float inv_sqrt_dh) {
#if defined(MMHA_USE_HMMA_FOR_REDUCTION) && defined(__CUDA_ARCH__) && \
    __CUDA_ARCH__ >= 750
#ifdef MMHA_USE_FP32_ACUM_FOR_FMA
  using K_vec_acum = typename K_vec_acum_fp32_<uint32_t>::Type;
#else
  using K_vec_acum = uint32_t;
#endif
  K_vec_acum inv_q = mul<K_vec_acum, uint32_t, float>(q[0], inv_sqrt_dh);
  K_vec_acum qk_vec = mul<K_vec_acum, K_vec_acum, uint32_t>(inv_q, k[0]);
#pragma unroll
  for (int ii = 1; ii < N; ++ii) {
    inv_q = mul<K_vec_acum, uint32_t, float>(q[ii], inv_sqrt_dh);
    qk_vec = fma(inv_q, k[ii], qk_vec);
  }
#ifdef MMHA_USE_FP32_ACUM_FOR_FMA
  uint32_t qk_vec_ = float2_to_half2(qk_vec);
  return hmma_fp32_tensorcore(make_uint2(qk_vec_, 0u), 0x3c003c00u).x;
#else
  return hmma_fp32_tensorcore(make_uint2(qk_vec, 0u), 0x3c003c00u).x;
#endif
#else
  return 0.f;
#endif
}

template <typename T, int THREADS_PER_KEY>
struct Qk_dot {
  template <typename K_vec, int N>
  static inline __device__ float dot(const K_vec (&q)[N],
                                     const K_vec (&k)[N],
                                     float inv_sqrt_dh) {
    return qk_dot_<THREADS_PER_KEY>(q, k, inv_sqrt_dh);
  }
};

template <>
struct Qk_dot<float16, 4> {
  template <int N>
  static inline __device__ float dot(const uint32_t (&q)[N],
                                     const uint32_t (&k)[N],
                                     float inv_sqrt_dh) {
#if defined(MMHA_USE_HMMA_FOR_REDUCTION) && defined(__CUDA_ARCH__) && \
    __CUDA_ARCH__ >= 750
    return qk_hmma_dot_(q, k, inv_sqrt_dh);
#else
    return qk_dot_<4>(q, k, inv_sqrt_dh);
#endif
  }
};

template <int WARPS_PER_BLOCK, int WARP_SIZE = 32>
inline __device__ float block_sum(float *red_smem, float sum) {
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;

#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
  }

  if (lane == 0) {
    red_smem[warp] = sum;
  }
  __syncthreads();

  if (lane < WARPS_PER_BLOCK) {
    sum = red_smem[lane];
  }

#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    sum += __shfl_xor_sync(uint32_t(-1), sum, mask);
  }

  return __shfl_sync(uint32_t(-1), sum, 0);
}

inline __device__ void convert_from_float(float &dst, float src) {  // NOLINT
  dst = src;
}

inline __device__ void convert_from_float(float4 &dst, float4 src) {  // NOLINT
  dst = src;
}

inline __device__ void convert_from_float(phi::float16 &dst,  // NOLINT
                                          float src) {
  dst = static_cast<phi::float16>(src);
}

inline __device__ void convert_from_float(uint4 &dst, Float8_ src) {  // NOLINT
  dst.x = float2_to_half2(src.x);
  dst.y = float2_to_half2(src.y);
  dst.z = float2_to_half2(src.z);
  dst.w = float2_to_half2(src.w);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
#ifdef ENABLE_BF16
inline __device__ void convert_from_float(__hip_bfloat16 &dst,  // NOLINT
                                          float src) {         // NOLINT
  dst = __float2bfloat16(src);
}

inline __device__ void convert_from_float(__hip_bfloat162 &dst,  // NOLINT
                                          float2 src) {         // NOLINT
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  dst = __float22bfloat162_rn(src);
#else
  dst = __floats2bfloat162_rn(src.x, src.y);
#endif
}

inline __device__ void convert_from_float(bf16_4_t &dst,  // NOLINT
                                          Float4_ src) {  // NOLINT
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  dst.x = __float22bfloat162_rn(src.x);
  dst.y = __float22bfloat162_rn(src.y);
#else
  dst.x = __floats2bfloat162_rn(src.x.x, src.x.y);
  dst.y = __floats2bfloat162_rn(src.y.x, src.y.y);
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ void convert_from_float(bf16_4_t &dst,  // NOLINT
                                          float4 src) {   // NOLINT
  convert_from_float(
      dst, Float4_{make_float2(src.x, src.y), make_float2(src.z, src.w)});
}

inline __device__ void convert_from_float(bf16_8_t &dst,  // NOLINT
                                          Float8_ src) {  // NOLINT
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  dst.x = __float22bfloat162_rn(src.x);
  dst.y = __float22bfloat162_rn(src.y);
  dst.z = __float22bfloat162_rn(src.z);
  dst.w = __float22bfloat162_rn(src.w);
#else
  dst.x = __floats2bfloat162_rn(src.x.x, src.x.y);
  dst.y = __floats2bfloat162_rn(src.y.x, src.y.y);
  dst.z = __floats2bfloat162_rn(src.z.x, src.z.y);
  dst.w = __floats2bfloat162_rn(src.w.x, src.w.y);
#endif
}
#endif  // ENABLE_BF16

////////////////////////////////////////////////////////////////////////////////////////////////////

inline __device__ void zero(uint16_t &dst) { dst = uint16_t(0); }  // NOLINT

template <typename T>
inline __device__ void zero(T &dst) {  // NOLINT
  constexpr int WORDS = sizeof(T) / 4;
  union {
    T raw;
    uint32_t words[WORDS];
  } tmp;
#pragma unroll
  for (int ii = 0; ii < WORDS; ++ii) {
    tmp.words[ii] = 0u;
  }
  dst = tmp.raw;
}

template <typename T,
          int Dh,
          int Dh_MAX,
          int THREADS_PER_KEY,
          int THREADS_PER_VALUE,
          int THREADS_PER_BLOCK,
          typename LoadFunc,
          typename StoreFunc>
__global__ void masked_multihead_attention_kernel(
    Masked_multihead_attention_params<T> params,
    LoadFunc load_func,
    StoreFunc store_func) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  const int bi = blockIdx.y;
  // params.sequence_lengths[bi] means how many k and v we have cached in cacke_kv
  if (params.sequence_lengths && params.sequence_lengths[bi] == 0 && 0) {
    return;
  }

//printf("%d blockDim.x \n", blockDim.x);
// printf("%d blockDim.y \n", blockDim.y);
// printf("%d blockDim.z \n", blockDim.z);

  typedef PDDataTypeTraits<T> traits_;
  typedef typename traits_::DataType DataType_;

  static_assert(Dh_MAX % THREADS_PER_KEY == 0, "");
  static_assert(Dh_MAX % THREADS_PER_VALUE == 0, "");

  constexpr int WARP_SIZE = 32;
  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  extern __shared__ char smem_[];

  float *qk_smem = reinterpret_cast<float *>(smem_);

  char *logits_smem_ = smem_;
  // fp32 accum for logits
  float *logits_smem = reinterpret_cast<float *>(logits_smem_);

  T *out_smem = reinterpret_cast<T *>(smem_);

  __shared__ float red_smem[WARPS_PER_BLOCK * 2];
  using Qk_vec = typename Qk_vec_<T, Dh_MAX>::Type;
  using Qk_vec_RoPE = typename Qk_vec_RoPE_<T, float, Dh_MAX>::Type;
  __shared__ __align__(sizeof(Qk_vec)) T q_smem[Dh_MAX];

  // beam id
  const int beami = bi % params.beam_width;
  // real batch id
  const int bbi = bi / params.beam_width;
  const int hi = blockIdx.x;
  const int bhi = bi * params.num_head + hi;

  const int kv_num_head = params.kv_num_head;
  const int num_head_per_group = params.num_head / kv_num_head;
  // hi means the head index in query processed by this cuda thread.
  // kv_bhi means the merged batch and head index in key and value processed by
  // this cuda thread.
  const int kv_bhi = bi * kv_num_head + hi / num_head_per_group;

  const int bbhi = bbi * params.beam_width * params.num_head + hi;
  const int ti =
      params.cum_offsets ? bi * params.seq_len - params.cum_offsets[bi] : -1;
  const int thi = params.cum_offsets ? ti * params.num_head + hi : -1;
  const int tid = threadIdx.x;

  const int bi_seq_len_offset = bi * params.max_seq_length;

  float qk_max = -FLT_MAX;
  float qk = 0;

  int act_time_step = params.sequence_lengths == nullptr
                          ? params.timestep
                          : params.sequence_lengths[bi];

  // qkv [B, S=1, num_head + 2 * kv_num_head, head_dim]
  // this hi means the head index in query!
  int qkv_base_offset = bi * (params.num_head + 2 * kv_num_head) * Dh + hi * Dh;

  constexpr int QK_VEC_SIZE = sizeof(Qk_vec) / sizeof(T);
  static_assert(Dh_MAX % QK_VEC_SIZE == 0, "");
  // Use block reduction if needed
  // static_assert(Dh_MAX / QK_VEC_SIZE <= WARP_SIZE, "");
  constexpr int QK_VECS_PER_WARP = Dh_MAX / QK_VEC_SIZE;

  // cache_k, [B, num_head, head_dim / x, max_seq_len, x]
  // x == 4/8 for FP32/FP16, 128bit, 16Byte
  constexpr int QK_ELTS_IN_16B = 16 / sizeof(T);
  constexpr int QK_VECS_IN_16B = 16 / sizeof(Qk_vec);

  // const T *q_base = params.qkv;
  // const T *k_base = params.qkv + params.num_head * Dh;
  T *q_bias_base = nullptr;
  T *k_bias_base = nullptr;

  if (params.add_qkv_bias) {
    q_bias_base = params.qkv_bias;
    k_bias_base = params.qkv_bias + params.num_head * Dh;
  }
  // QK_VECS_PER_WARP 表示每个
  if (tid < QK_VECS_PER_WARP) {
    int qk_offset = qkv_base_offset + tid * QK_VEC_SIZE;
    int q_bias_offset = hi * Dh + tid * QK_VEC_SIZE;
    int k_bias_offset = hi / num_head_per_group * Dh + tid * QK_VEC_SIZE;

    Qk_vec q;
    zero(q);
    // q = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
    //         ? *reinterpret_cast<const Qk_vec *>(&q_base[qk_offset])
    //         : q;
    if (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh) {
      load_func.template load<Qk_vec>(q, qk_offset);
    }

    Qk_vec k;
    zero(k);
    // k = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
    //         ? *reinterpret_cast<const Qk_vec *>(&k_base[qk_offset])
    //         : k;
    if (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh) {
      load_func.template load<Qk_vec>(k,
                                      params.num_head * Dh + qk_offset -
                                          hi * Dh +
                                          hi / num_head_per_group * Dh);
    }

    if (params.add_qkv_bias) {
      Qk_vec q_bias;
      zero(q_bias);
      Qk_vec k_bias;
      zero(k_bias);

      q_bias =
          (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
              ? *reinterpret_cast<const Qk_vec *>(&q_bias_base[q_bias_offset])
              : q_bias;
      k_bias =
          (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
              ? *reinterpret_cast<const Qk_vec *>(&k_bias_base[k_bias_offset])
              : k_bias;

      q = add(q, q_bias);
      // TODO(wangxi): See this https://github.com/microsoft/unilm/issues/510
      //   we may not require k_bias.
      k = add(k, k_bias);
    }

    if (!params.neox_rotary_style) {
      if (params.rotary_emb_dims != 0) {
        int rotary_offset = bi * Dh + tid * QK_VEC_SIZE;
        const float *cos_base = params.rotary_emb;
        const float *sin_base = params.rotary_emb + params.batch_size * Dh;
        Qk_vec_RoPE cos_emb, sin_emb;
        zero(cos_emb);
        zero(sin_emb);
        cos_emb = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
                      ? *reinterpret_cast<const Qk_vec_RoPE *>(
                            &cos_base[rotary_offset])
                      : cos_emb;
        sin_emb = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
                      ? *reinterpret_cast<const Qk_vec_RoPE *>(
                            &sin_base[rotary_offset])
                      : sin_emb;
        apply_rotary_embedding(q, k, cos_emb, sin_emb);
      }
    } else {
      /* old rotary pos emb */
      if (params.rotary_emb_dims != 0) {
        int last_dim = Dh / params.rotary_emb_dims;
        int half_lastdim = last_dim / 2;
        int rotary_offset = bi * Dh + tid * QK_VEC_SIZE;
        const float *cos_base = params.rotary_emb;
        const float *sin_base = params.rotary_emb + params.batch_size * Dh;
        int stride = half_lastdim / QK_VEC_SIZE;
        int stride_all_lastdim = 2 * stride;
        int right_id = tid / stride_all_lastdim * stride_all_lastdim +
                       (tid + stride) % (stride_all_lastdim);
        int qk_right_offset = qkv_base_offset + right_id * QK_VEC_SIZE;
        int q_right_bias_offset = hi * Dh + right_id * QK_VEC_SIZE;
        int k_right_bias_offset =
            hi / num_head_per_group * Dh + right_id * QK_VEC_SIZE;
        Qk_vec q_right;
        zero(q_right);
        // q_right =
        //     (Dh == Dh_MAX || right_id * QK_VEC_SIZE < Dh)
        //         ? *reinterpret_cast<const Qk_vec *>(&q_base[qk_right_offset])
        //         : q_right;
        if (Dh == Dh_MAX || right_id * QK_VEC_SIZE < Dh) {
          load_func.template load<Qk_vec>(q_right, qk_right_offset);
        }
        Qk_vec k_right;
        zero(k_right);
        // k_right =
        //     (Dh == Dh_MAX || right_id * QK_VEC_SIZE < Dh)
        //         ? *reinterpret_cast<const Qk_vec *>(&k_base[qk_right_offset])
        //         : k_right;
        if (Dh == Dh_MAX || right_id * QK_VEC_SIZE < Dh) {
          load_func.template load<Qk_vec>(k_right,
                                          params.num_head * Dh +
                                              qk_right_offset - hi * Dh +
                                              hi / num_head_per_group * Dh);
        }

        if (params.add_qkv_bias) {
          Qk_vec q_right_bias;
          zero(q_right_bias);
          q_right_bias = (Dh == Dh_MAX || right_id * QK_VEC_SIZE < Dh)
                             ? *reinterpret_cast<const Qk_vec *>(
                                   &q_bias_base[q_right_bias_offset])
                             : q_right_bias;
          Qk_vec k_right_bias;
          zero(k_right_bias);
          k_right_bias = (Dh == Dh_MAX || right_id * QK_VEC_SIZE < Dh)
                             ? *reinterpret_cast<const Qk_vec *>(
                                   &k_bias_base[k_right_bias_offset])
                             : k_right_bias;

          q_right = add(q_right, q_right_bias);
          k_right = add(k_right, k_right_bias);
        }

        Qk_vec_RoPE cos_emb;
        zero(cos_emb);
        cos_emb = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
                      ? *reinterpret_cast<const Qk_vec_RoPE *>(
                            &cos_base[rotary_offset])
                      : cos_emb;

        Qk_vec_RoPE sin_emb;
        zero(sin_emb);
        sin_emb = (Dh == Dh_MAX || tid * QK_VEC_SIZE < Dh)
                      ? *reinterpret_cast<const Qk_vec_RoPE *>(
                            &sin_base[rotary_offset])
                      : sin_emb;
        float alpha = (tid % stride_all_lastdim) < stride
                          ? static_cast<float>(-1)
                          : static_cast<float>(1);
        q = apply_rotary_emb<Qk_vec, Qk_vec_RoPE>(
            q, q_right, cos_emb, sin_emb, alpha);
        k = apply_rotary_emb<Qk_vec, Qk_vec_RoPE>(
            k, k_right, cos_emb, sin_emb, alpha);
      }
    }

    *reinterpret_cast<Qk_vec *>(&q_smem[tid * QK_VEC_SIZE]) = q;

    int co = tid / QK_VECS_IN_16B;
    int ci = (tid % QK_VECS_IN_16B) * QK_VEC_SIZE;
    int offset = kv_bhi * params.max_seq_length * Dh +
                 co * params.max_seq_length * QK_ELTS_IN_16B +
                 act_time_step * QK_ELTS_IN_16B + ci;
    if (Dh == Dh_MAX || co < Dh / QK_ELTS_IN_16B) {
      *reinterpret_cast<Qk_vec *>(&params.cache_kv[offset]) = k;
    }

    qk = dot<Qk_vec, Qk_vec>(q, k);

    if (QK_VECS_PER_WARP <= WARP_SIZE) {
#pragma unroll
      for (int mask = QK_VECS_PER_WARP / 2; mask >= 1; mask /= 2) {
        qk += __shfl_xor_sync(shfl_mask(QK_VECS_PER_WARP), qk, mask);
      }
    }
  }
  if (QK_VECS_PER_WARP > WARP_SIZE) {
    constexpr int WARPS_PER_RED =
        (QK_VECS_PER_WARP + WARP_SIZE - 1) / WARP_SIZE;
    qk = block_sum<WARPS_PER_RED>(&red_smem[WARPS_PER_RED], qk);
  }
  if (tid == 0) {
    // NOTE(wangxi): mask must be 0.0
    // T mask = params.attn_mask[
    //    bi * (params.timestep + 1) + params.timestep];
    // qk += static_cast<float>(mask);
    qk *= params.inv_sqrt_dh;
    if (params.attn_mask) {
      auto mask_bhi = params.mask_broadcast_num_heads ? bi : bhi;
      T mask = params.attn_mask[mask_bhi * params.mask_length + act_time_step];
      qk += static_cast<float>(mask);
    }
    qk_max = qk;
    qk_smem[act_time_step] = qk;
  }
  __syncthreads();

  using K_vec = typename K_vec_<T, THREADS_PER_KEY>::Type;
  constexpr int K_VEC_SIZE = sizeof(K_vec) / sizeof(T);
  static_assert(Dh_MAX % K_VEC_SIZE == 0, "");
  constexpr int K_ELTS_PER_THREAD = Dh_MAX / THREADS_PER_KEY;
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;

  int ko = tid / THREADS_PER_KEY;
  int ki = (tid % THREADS_PER_KEY) * K_VEC_SIZE;

  static_assert(Dh_MAX == THREADS_PER_KEY * K_VEC_SIZE * K_VECS_PER_THREAD, "");

  K_vec q[K_VECS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < K_VECS_PER_THREAD; ++i) {
    q[i] = *reinterpret_cast<const K_vec *>(
        &q_smem[ki + i * THREADS_PER_KEY * K_VEC_SIZE]);
  }

  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  T *k_cache = &params.cache_kv[kv_bhi * params.max_seq_length * Dh + ki];
  T *k_cache_batch = &params.cache_kv[bbhi * params.max_seq_length * Dh + ki];
  int ti_end = div_up(act_time_step, K_PER_WARP) * K_PER_WARP;

  const int *beam_offsets = params.beam_cache_offset
                                ? &params.beam_cache_offset[bi_seq_len_offset]
                                : nullptr;
  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    const int beam_offset = beam_offsets ? beam_offsets[ti] * params.num_head *
                                               params.max_seq_length * Dh
                                         : 0;
    K_vec k[K_VECS_PER_THREAD];
    K_vec k_vec_zero;
    zero(k_vec_zero);
#pragma unroll
    for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
      int jj = ii * params.max_seq_length + ti;
      if (ti < act_time_step) {
        if (beam_offset) {
          k[ii] =
              (Dh == Dh_MAX || jj * QK_ELTS_IN_16B < Dh * params.max_seq_length)
                  ? *reinterpret_cast<const K_vec *>(
                        &k_cache_batch[beam_offset + jj * QK_ELTS_IN_16B])
                  : k_vec_zero;
        } else {
          k[ii] =
              (Dh == Dh_MAX || jj * QK_ELTS_IN_16B < Dh * params.max_seq_length)
                  ? *reinterpret_cast<const K_vec *>(
                        &k_cache[jj * QK_ELTS_IN_16B])
                  : k_vec_zero;
        }
      }
    }

    // NOTE(liyurui): We should multiple q with inv_sqrt_dh first, for dot(q, k)
    // may overflow with FP16 in large model.
    float qk = Qk_dot<T, THREADS_PER_KEY>::dot(q, k, params.inv_sqrt_dh);

    // bool is_mask = false;
    if (ti < act_time_step && tid % THREADS_PER_KEY == 0) {
      // qk_max = is_mask ? qk_max : fmaxf(qk_max, qk);
      auto mask_bhi = params.mask_broadcast_num_heads ? bi : bhi;
      // T mask = params.attn_mask[mask_bhi * (params.timestep + 1) + ti];
      if (params.attn_mask) {
        T mask = params.attn_mask[mask_bhi * params.mask_length + ti];
        qk += static_cast<float>(mask);
      }
      qk_max = fmaxf(qk_max, qk);

      qk_smem[ti] = qk;
    }
  }

#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  const int warp = tid / WARP_SIZE;
  const int lane = tid % WARP_SIZE;

  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  __syncthreads();

  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  float sum = 0.f;
  for (int ti = tid; ti <= act_time_step; ti += THREADS_PER_BLOCK) {
    // bool is_mask = false;
    // float logit = is_mask ? 0.f : __expf(qk_smem[ti] - qk_max);
    float logit = __expf(qk_smem[ti] - qk_max);
    sum += logit;
    qk_smem[ti] = logit;
  }

  sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], sum);

  // FIXME(wangxi): need add 1.e-6f?
  float inv_sum = __fdividef(1.f, sum + 1.e-6f);

  for (int ti = tid; ti <= act_time_step; ti += THREADS_PER_BLOCK) {
    convert_from_float(logits_smem[ti], qk_smem[ti] * inv_sum);
  }
  __syncthreads();

  constexpr int V_VEC_SIZE = Dh_MAX / THREADS_PER_VALUE;
  using V_vec = typename V_vec_<T, V_VEC_SIZE>::Type;



  // now we have got [1, seq] ，distributed in logits_smem.
  // next we compute [1, seq] * [seq, head_dim] = [1, head_dim] using all threads in a thread block.
  // threads in a thread block in rearranged as [THREADS_PER_VALUE, THREADS_PER_BLOCK / THREADS_PER_VALUE]
  // THREADS_PER_VALUE is contiguous .
  // THREADS_PER_VALUE means num of threads per value's head_dim.
  // we split the seq dimension for more cuda threads to compute.
  // vo means the first seq index processed by this cuda thread in the value.
  // vi means the head_dim index processed by this cuda thread in the value.
  // so this cuda thread compute [1, k] * [k, vi:vi+V_VEC_SIZE] and k starts
  // from vo and increases by a step V_PER_ITER.

  // 每个cuda thread计算 [1, K] * [K , vi:vi+V_VEC_SIZE]
  // 但是K不是连续的 and trided by V_PER_ITER! and begins at vo.
  int vo = tid / THREADS_PER_VALUE;
  // 最后就是只有 vo == 0 的那些cuda thread，才需要写入到输出中！
  int vi = (tid % THREADS_PER_VALUE) * V_VEC_SIZE;

  T *v_cache = &params.cache_kv[params.cache_batch_size * kv_num_head *
                                    params.max_seq_length * Dh +
                                kv_bhi * params.max_seq_length * Dh + vi];
  T *v_cache_batch = &params.cache_kv[params.batch_size * params.num_head *
                                          params.max_seq_length * Dh +
                                      bbhi * params.max_seq_length * Dh + vi];

#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
  using V_vec_acum = typename V_vec_acum_fp32_<V_vec>::Type;
#else
  using V_vec_acum = V_vec;
#endif

  V_vec_acum out;
  zero(out);
  // V_PER_ITER is used to strip-mined the seq dimension.
  constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;
  if (Dh == Dh_MAX || vi < Dh) {
    for (int ti = vo; ti < act_time_step; ti += V_PER_ITER) {
      const int beam_offset =
          beam_offsets
              ? beam_offsets[ti] * params.num_head * params.max_seq_length * Dh
              : 0;
      V_vec v;
      if (beam_offset) {
        v = *reinterpret_cast<const V_vec *>(
            &v_cache_batch[beam_offset + ti * Dh]);
      } else {
        v = *reinterpret_cast<const V_vec *>(&v_cache[ti * Dh]);
      }
#if defined(MMHA_USE_FP32_ACUM_FOR_LOGITS)
      float logit = logits_smem[ti];
      out = fma(logit, cast_to_float(v), out);
#else
      DataType_ logit = static_cast<DataType_>(logits_smem[ti]);
      // Update the partial sums.
      out = fma(logit, v, out);
#endif
    }
  }

  V_vec v_bias;
  zero(v_bias);
  // now we process the last v.
  // 为啥不让vo等于0的那些cuda thread写入呢？？
  if (vo == (act_time_step % V_PER_ITER * 0) && (Dh == Dh_MAX || vi < Dh)) {
    // V_vec v = *reinterpret_cast<const V_vec *>(
    //     &params.qkv[2 * params.num_head * Dh + qkv_base_offset + vi]);
    V_vec v;
    load_func.template load<V_vec>(v,
                                   qkv_base_offset + vi - hi * Dh +
                                       params.num_head * Dh + kv_num_head * Dh +
                                       hi / num_head_per_group * Dh);
    if (params.add_qkv_bias) {
      v_bias = *reinterpret_cast<const V_vec *>(
          &params.qkv_bias[(kv_num_head + params.num_head) * Dh +
                           hi / num_head_per_group * Dh + vi]);
      v = add(v, v_bias);
    }

    // if (tid == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
    //   half* tmp = (half*)(&v);
    //   printf("%f \n", (float)(*(tmp + 0)));
    //   printf("%f \n", (float)(*(tmp + 1)));
    //   printf("%f \n", (float)(*(tmp + 2)));
    //   printf("%f \n", (float)(*(tmp + 3)));
    // }

    *reinterpret_cast<V_vec *>(&v_cache[act_time_step * Dh]) = v;

#if defined(MMHA_USE_FP32_ACUM_FOR_LOGITS)
    out = fma(logits_smem[act_time_step], cast_to_float(v), out);
#else
    out = fma(logits_smem[act_time_step], v, out);
#endif
  }

  __syncthreads();

  // if (tid == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
  //   float* tmp = (float*)(&out);
  //   printf("%f \n", (float)(*(tmp + 0)));
  //   printf("%f \n", (float)(*(tmp + 1)));
  //   printf("%f \n", (float)(*(tmp + 2)));
  // }
  // 将每 V_PER_ITER 个cuda thread进行规约！

  // 现在把每个cuda thread里面的out进行规约！
  // now we do the reduction in the seq dimension to get [1, head_dim].
  if (Dh == Dh_MAX || vi < Dh) {
#pragma unroll
    for (int active_groups = V_PER_ITER; active_groups >= 2;
         active_groups /= 2) {
      int midpoint = active_groups / 2;

      if (vo >= midpoint && vo < active_groups && (Dh == Dh_MAX || vi < Dh)) {
#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
        convert_from_float(
            *reinterpret_cast<V_vec *>(&out_smem[(vo - midpoint) * Dh + vi]),
            out);
#else
        *reinterpret_cast<V_vec *>(&out_smem[(vo - midpoint) * Dh + vi]) = out;
#endif
      }
      __syncthreads();
      if (vo < midpoint && (Dh == Dh_MAX || vi < Dh)) {
        out =
            add(*reinterpret_cast<const V_vec *>(&out_smem[vo * Dh + vi]), out);
      }
      __syncthreads();
    }
  }
  // 最后就是只有 vo == 0 的那些cuda thread，才需要写入到输出中！

  // write the [1, head_dim] result back to global memory.
  if (vo == 0 && (Dh == Dh_MAX || vi < Dh)) {
#ifdef MMHA_USE_FP32_ACUM_FOR_OUT
    // convert_from_float(*reinterpret_cast<V_vec *>(&params.out[bhi * Dh +
    // vi]),
    //                    out);
    V_vec tmp_out;
    convert_from_float(tmp_out, out);
    store_func.template store<V_vec>(tmp_out,
                                     thi != -1 ? thi * Dh + vi : bhi * Dh + vi);
#else
    // *reinterpret_cast<V_vec *>(&params.out[bhi * Dh + vi]) = out;
    store_func.template store<V_vec>(out,
                                     thi != -1 ? thi * Dh + vi : bhi * Dh + vi);
#endif
  }

#else
  assert(false);
#endif
}

template <typename T>
inline size_t smem_size_in_bytes(
    const Masked_multihead_attention_params<T> &params,
    int dim_head,
    int threads_per_value,
    int threads_per_block) {
  size_t qk_sz = div_up(params.timestep + 1, 4) * 16;
  size_t logits_sz = 0;

#ifndef MMHA_USE_FP32_ACUM_FOR_LOGITS  // NOLINT
  if (sizeof(T) != 4) {
    logits_sz = div_up(params.max_seq_length, 4) * 4 * sizeof(T);
  }
#endif  // NOLINT
  size_t softmax_sz = qk_sz + logits_sz;

  int rows_per_red = threads_per_block / threads_per_value;
  size_t red_sz = rows_per_red * dim_head * sizeof(T) / 2;

  return max(softmax_sz, red_sz);
}

#define MMHA_LAUNCH_KERNEL(T,                                             \
                           Dh,                                            \
                           Dh_MAX,                                        \
                           THDS_PER_KEY,                                  \
                           THDS_PER_VALUE,                                \
                           THDS_PER_BLOCK,                                \
                           stream,                                        \
                           load_func,                                     \
                           store_func)                                    \
  size_t smem_sz =                                                        \
      smem_size_in_bytes<T>(params, Dh, THDS_PER_VALUE, THDS_PER_BLOCK);  \
  constexpr auto kernel_fn =                                              \
      masked_multihead_attention_kernel<T,                                \
                                        Dh,                               \
                                        Dh_MAX,                           \
                                        THDS_PER_KEY,                     \
                                        THDS_PER_VALUE,                   \
                                        THDS_PER_BLOCK,                   \
                                        decltype(load_func),              \
                                        decltype(store_func)>;            \
  if (smem_sz > 0xc000) {                                                 \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                 \
        kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_sz); \
  }                                                                       \
  dim3 grid(params.num_head, params.batch_size);                          \
  kernel_fn<<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(                   \
      params, load_func, store_func)

template <typename T, int Dh, int Dh_MAX, typename LoadFunc, typename StoreFunc>
void fmha_launch_kernel(const Masked_multihead_attention_params<T> &params,
                        const hipStream_t &stream,
                        LoadFunc load_func,
                        StoreFunc store_func) {
  constexpr int THREADS_PER_VALUE = Dh_MAX * sizeof(T) / 16;
  if (params.timestep < 32) {
    MMHA_LAUNCH_KERNEL(
        T, Dh, Dh_MAX, 4, THREADS_PER_VALUE, 64, stream, load_func, store_func);
  } else if (params.timestep < 2048) {
#if defined(MMHA_USE_HMMA_FOR_REDUCTION) && defined(__CUDA_ARCH__) && \
    __CUDA_ARCH__ >= 750
    MMHA_LAUNCH_KERNEL(T,
                       Dh,
                       Dh_MAX,
                       4,
                       THREADS_PER_VALUE,
                       256,
                       stream,
                       load_func,
                       store_func);
#else
    MMHA_LAUNCH_KERNEL(T,
                       Dh,
                       Dh_MAX,
                       2,
                       THREADS_PER_VALUE,
                       128,
                       stream,
                       load_func,
                       store_func);
#endif
  } else {
    MMHA_LAUNCH_KERNEL(T,
                       Dh,
                       Dh_MAX,
                       1,
                       THREADS_PER_VALUE,
                       256,
                       stream,
                       load_func,
                       store_func);
  }
}

template <typename T, typename LoadFunc, typename StoreFunc>
void fmha_impl(const phi::GPUContext &dev_ctx,
               const Masked_multihead_attention_params<T> &params,
               int dim_head,
               LoadFunc load_func,
               StoreFunc store_func) {
  switch (dim_head) {
    case 10:
      fmha_launch_kernel<T, 10, 32>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 16:
      fmha_launch_kernel<T, 16, 32>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 26:
      fmha_launch_kernel<T, 26, 32>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 32:
      fmha_launch_kernel<T, 32, 32>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 64:
      fmha_launch_kernel<T, 64, 64>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    // for opt model
    case 80:
      fmha_launch_kernel<T, 80, 128>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 96:
      fmha_launch_kernel<T, 96, 128>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 128:
      fmha_launch_kernel<T, 128, 128>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    case 192:
      fmha_launch_kernel<T, 192, 256>(
          params, dev_ctx.stream(), load_func, store_func);
      break;
    default:
      PADDLE_THROW(
          phi::errors::Unimplemented("Dim_head = %d is unsupport!", dim_head));
  }
}

template <typename T, typename LoadT = T>
struct MMHALoad {
  explicit MMHALoad(const LoadT *src) : src_(src) {}

  template <typename Vec>
  __device__ void load(Vec &dst, int idx) {
    dst = *reinterpret_cast<const Vec *>(src_ + idx);
  }

  const LoadT *src_;
};

template <typename T, typename StoreT = T, bool Smooth = false>
struct MMHAStore {
  explicit MMHAStore(StoreT *dst) : dst_(dst) {}

  template <typename Vec>
  __device__ void store(Vec &src, int idx) {
    *reinterpret_cast<Vec *>(dst_ + idx) = src;
  }

  StoreT *dst_;
};

template <typename T>
struct MMHAStore<T, T, true> {
  MMHAStore(T *dst, const T *shift, const T *smooth, const int cols)
      : dst_(dst), shift_(shift), smooth_(smooth), cols_(cols) {}

  template <typename Vec>
  __device__ void store(Vec &src, int idx) {
    constexpr int VecSize = sizeof(Vec) / sizeof(T);
    using TVec = phi::AlignedVector<T, VecSize>;
    TVec src_vec;
    TVec shift_vec;
    TVec smooth_vec;

    *reinterpret_cast<Vec *>(&src_vec) = src;
    phi::Load<T, VecSize>(shift_ + idx % cols_, &shift_vec);
    phi::Load<T, VecSize>(smooth_ + idx % cols_, &smooth_vec);

#pragma unroll
    for (int i = 0; i < VecSize; i++) {
      src_vec[i] = (src_vec[i] + shift_vec[i]) * smooth_vec[i];
    }

    phi::Store<T, VecSize>(src_vec, dst_ + idx);
  }

  T *dst_;
  const T *shift_;
  const T *smooth_;
  const int cols_;
};

template <typename T>
struct MMHALoad<T, int32_t> {
  MMHALoad(const int32_t *src, const float *dequant_scales, const int cols)
      : src_(src), dequant_scales_(dequant_scales), cols_(cols) {}

  template <typename Vec>
  __device__ void load(Vec &dst, int idx) {
    constexpr int VecSize = sizeof(Vec) / sizeof(T);
    using SrcVec = phi::AlignedVector<int32_t, VecSize>;
    using DstVec = phi::AlignedVector<T, VecSize>;
    using ScaleVec = phi::AlignedVector<float, VecSize>;

    SrcVec src_vec;
    DstVec dst_vec;
    ScaleVec scale_vec;

    phi::Load<int32_t, VecSize>(src_ + idx, &src_vec);
    phi::Load<float, VecSize>(dequant_scales_ + idx % cols_, &scale_vec);
#pragma unroll
    for (int i = 0; i < VecSize; i++) {
      dst_vec[i] =
          static_cast<T>(static_cast<float>(src_vec[i]) * scale_vec[i]);
    }
    dst = *reinterpret_cast<Vec *>(&dst_vec);
  }

  const int32_t *src_;
  const float *dequant_scales_;
  const int cols_;
};

template <typename T>
struct MMHAStore<T, int8_t> {
  MMHAStore(int8_t *dst,
            const int quant_round_type,
            const float quant_scale,
            const float quant_max_bound,
            const float quant_min_bound)
      : dst_(dst),
        quant_round_type_(quant_round_type),
        quant_scale_(quant_scale),
        quant_max_bound_(quant_max_bound),
        quant_min_bound_(quant_min_bound) {}

  template <typename Vec>
  __device__ void store(Vec &src, int idx) {  // NOLINT
    constexpr int VecSize = sizeof(Vec) / sizeof(T);
    using SrcVec = phi::AlignedVector<T, VecSize>;
    using DstVec = phi::AlignedVector<int8_t, VecSize>;

    SrcVec src_vec;
    *reinterpret_cast<Vec *>(&src_vec) = src;

    DstVec dst_vec;
#pragma unroll
    for (int i = 0; i < VecSize; i++) {
      dst_vec[i] =
          QuantHelperFunc<float, int8_t>(static_cast<float>(src_vec[i]),
                                         quant_scale_,
                                         quant_round_type_,
                                         quant_max_bound_,
                                         quant_min_bound_);
    }

    phi::Store<int8_t, VecSize>(dst_vec, dst_ + idx);
  }

  int8_t *dst_;
  const int quant_round_type_;
  const float quant_scale_;
  const float quant_max_bound_;
  const float quant_min_bound_;
};

template <typename T>
struct MMHAStore<T, int8_t, true> {
  MMHAStore(int8_t *dst,
            const T *shift,
            const T *smooth,
            const int cols,
            const int quant_round_type,
            const float quant_scale,
            const float quant_max_bound,
            const float quant_min_bound)
      : dst_(dst),
        quant_round_type_(quant_round_type),
        quant_scale_(quant_scale),
        quant_max_bound_(quant_max_bound),
        quant_min_bound_(quant_min_bound),
        shift_(shift),
        smooth_(smooth),
        cols_(cols) {}

  template <typename Vec>
  __device__ void store(Vec &src, int idx) {  // NOLINT
    constexpr int VecSize = sizeof(Vec) / sizeof(T);
    using SrcVec = phi::AlignedVector<T, VecSize>;
    using DstVec = phi::AlignedVector<int8_t, VecSize>;

    SrcVec src_vec;
    DstVec dst_vec;
    SrcVec shift_vec;
    SrcVec smooth_vec;

    *reinterpret_cast<Vec *>(&src_vec) = src;
    phi::Load<T, VecSize>(shift_ + idx % cols_, &shift_vec);
    phi::Load<T, VecSize>(smooth_ + idx % cols_, &smooth_vec);

#pragma unroll
    for (int i = 0; i < VecSize; i++) {
      src_vec[i] = (src_vec[i] + shift_vec[i]) * smooth_vec[i];
      dst_vec[i] =
          QuantHelperFunc<float, int8_t>(static_cast<float>(src_vec[i]),
                                         quant_scale_,
                                         quant_round_type_,
                                         quant_max_bound_,
                                         quant_min_bound_);
    }

    phi::Store<int8_t, VecSize>(dst_vec, dst_ + idx);
  }

  int8_t *dst_;
  const T *shift_;
  const T *smooth_;
  const int cols_;
  const int quant_round_type_;
  const float quant_scale_;
  const float quant_max_bound_;
  const float quant_min_bound_;
};

template <typename T>
void DispatchFMHA(const phi::GPUContext &dev_ctx,
                  const phi::DenseTensor &qkv_tensor,
                  const Masked_multihead_attention_params<T> &params,
                  int num_head,
                  int dim_head,
                  phi::DenseTensor *out_tensor,
                  const phi::DenseTensor *dequant_qkv_scales = nullptr,
                  const float quant_fmha_out_scale = -1,
                  const int quant_round_type = 1,
                  const float quant_max_bound = 127.0f,
                  const float quant_min_bound = -127.0f) {
  if (dequant_qkv_scales != nullptr && quant_fmha_out_scale > 0) {
    MMHALoad<T, int32_t> load_func(qkv_tensor.data<int32_t>(),
                                   dequant_qkv_scales->data<float>(),
                                   3 * num_head * dim_head);
    MMHAStore<T, int8_t> store_func(out_tensor->data<int8_t>(),
                                    quant_round_type,
                                    quant_fmha_out_scale,
                                    quant_max_bound,
                                    quant_min_bound);
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  } else if (dequant_qkv_scales == nullptr && quant_fmha_out_scale > 0) {
    MMHALoad<T> load_func(qkv_tensor.data<T>());
    MMHAStore<T, int8_t> store_func(out_tensor->data<int8_t>(),
                                    quant_round_type,
                                    quant_fmha_out_scale,
                                    quant_max_bound,
                                    quant_min_bound);
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  } else if (dequant_qkv_scales != nullptr && quant_fmha_out_scale <= 0) {
    MMHALoad<T, int32_t> load_func(qkv_tensor.data<int32_t>(),
                                   dequant_qkv_scales->data<float>(),
                                   3 * num_head * dim_head);
    MMHAStore<T> store_func(out_tensor->data<T>());
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  } else {
    MMHALoad<T> load_func(qkv_tensor.data<T>());
    MMHAStore<T> store_func(out_tensor->data<T>());
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  }
}

template <typename T>
void DispatchFMHA(const phi::GPUContext &dev_ctx,
                  const phi::DenseTensor &qkv_tensor,
                  const phi::DenseTensor &shift,
                  const phi::DenseTensor &smooth,
                  const Masked_multihead_attention_params<T> &params,
                  int num_head,
                  int dim_head,
                  phi::DenseTensor *out_tensor,
                  const phi::DenseTensor *dequant_qkv_scales = nullptr,
                  const float quant_fmha_out_scale = -1,
                  const int quant_round_type = 1,
                  const float quant_max_bound = 127.0f,
                  const float quant_min_bound = -127.0f) {
  if (dequant_qkv_scales != nullptr && quant_fmha_out_scale > 0) {
    MMHALoad<T, int32_t> load_func(qkv_tensor.data<int32_t>(),
                                   dequant_qkv_scales->data<float>(),
                                   3 * num_head * dim_head);
    MMHAStore<T, int8_t, true> store_func(out_tensor->data<int8_t>(),
                                          shift.data<T>(),
                                          smooth.data<T>(),
                                          num_head * dim_head,
                                          quant_round_type,
                                          quant_fmha_out_scale,
                                          quant_max_bound,
                                          quant_min_bound);
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  } else if (dequant_qkv_scales == nullptr && quant_fmha_out_scale > 0) {
    MMHALoad<T> load_func(qkv_tensor.data<T>());
    MMHAStore<T, int8_t, true> store_func(out_tensor->data<int8_t>(),
                                          shift.data<T>(),
                                          smooth.data<T>(),
                                          num_head * dim_head,
                                          quant_round_type,
                                          quant_fmha_out_scale,
                                          quant_max_bound,
                                          quant_min_bound);
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  } else if (dequant_qkv_scales != nullptr && quant_fmha_out_scale <= 0) {
    MMHALoad<T, int32_t> load_func(qkv_tensor.data<int32_t>(),
                                   dequant_qkv_scales->data<float>(),
                                   3 * num_head * dim_head);
    MMHAStore<T, T, true> store_func(out_tensor->data<T>(),
                                     shift.data<T>(),
                                     smooth.data<T>(),
                                     num_head * dim_head);
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  } else {
    MMHALoad<T> load_func(qkv_tensor.data<T>());
    MMHAStore<T, T, true> store_func(out_tensor->data<T>(),
                                     shift.data<T>(),
                                     smooth.data<T>(),
                                     num_head * dim_head);
    fmha_impl(dev_ctx, params, dim_head, load_func, store_func);
  }
}

struct NormalVersion {};
struct UnusedVersion {};

template <typename T>
struct DispatchDtypeTrait {
  using FuncVersion = NormalVersion;
};

template <>
struct DispatchDtypeTrait<int32_t> {
  using FuncVersion = UnusedVersion;
};

template <typename T, typename Context>
void DispatchWithDtype(const Context &dev_ctx,
                       const DenseTensor &x,
                       const DenseTensor &cache_kv,
                       const paddle::optional<DenseTensor> &bias,
                       const paddle::optional<DenseTensor> &src_mask,
                       const paddle::optional<DenseTensor> &cum_offsets,
                       const paddle::optional<DenseTensor> &sequence_lengths,
                       const paddle::optional<DenseTensor> &rotary_tensor,
                       const paddle::optional<DenseTensor> &beam_cache_offset,
                       const paddle::optional<DenseTensor> &qkv_out_scale,
                       const paddle::optional<DenseTensor> &out_shift,
                       const paddle::optional<DenseTensor> &out_smooth,
                       int seq_len,
                       int rotary_emb_dims,
                       const bool use_neox_rotary_style,
                       const float out_scale,
                       const int quant_round_type,
                       const float quant_max_bound,
                       const float quant_min_bound,
                       DenseTensor *out,
                       DenseTensor *cache_kv_out,
                       DenseTensor *beam_cache_offset_out,
                       NormalVersion) {
  const auto &x_dims = x.dims();
  int bsz = x_dims[0];
  int cache_bsz = cache_kv.dims()[1];
  int max_seq_len = cache_kv.dims()[3];
  int dim_head = cache_kv.dims()[4];
  int timestep = max_seq_len;
  float inv_sqrt_dh = 1. / sqrt(dim_head);

  int k_num_head = cache_kv.dims()[2];
  int v_num_head = k_num_head;
  // this num_head means query's head
  int num_head =
      x.dims()[x.dims().size() - 1] / dim_head - k_num_head - v_num_head;

  Masked_multihead_attention_params<T> params;
  bool mask_broadcast_num_heads = true;

  params.add_qkv_bias = false;
  if (bias) {
    params.add_qkv_bias = true;
    params.qkv_bias = const_cast<T *>(bias->data<T>());
  }

  if (src_mask) {
    if (src_mask->dims()[1] == 1) {
      mask_broadcast_num_heads = true;
    } else if (src_mask->dims()[1] == num_head) {
      mask_broadcast_num_heads = false;
    } else {
      PADDLE_THROW(errors::InvalidArgument(
          "Unknow dimension for attn_mask, the num_head(2nd) "
          "dimension is invalid, it should be 1 or num_head(%d), "
          "but got %d",
          num_head,
          src_mask->dims()[1]));
    }
    params.attn_mask = src_mask->data<T>();
    params.mask_length = src_mask->dims()[3];
    timestep = src_mask->dims()[3] - 1;
  }

  if (out_scale > 0) {
    dev_ctx.template Alloc<int8_t>(out);
  } else {
    dev_ctx.template Alloc<T>(out);
  }

  if (sequence_lengths) {
    params.sequence_lengths = sequence_lengths->data<int>();
  }

  if (cum_offsets) {
    params.cum_offsets = cum_offsets->data<int>();
  } else {
    params.cum_offsets = nullptr;
  }

  if (rotary_emb_dims > 0) {
    params.rotary_emb = rotary_tensor->data<float>();
  } else {
    params.rotary_emb = nullptr;
  }

  if (beam_cache_offset) {
    params.beam_cache_offset = beam_cache_offset->data<int>();
    params.beam_width = beam_cache_offset->dims()[1];
  }

  params.mask_broadcast_num_heads = mask_broadcast_num_heads;
  params.cache_kv = const_cast<T *>(cache_kv_out->data<T>());
  params.neox_rotary_style = use_neox_rotary_style;
  params.batch_size = bsz;
  params.cache_batch_size = cache_bsz;
  params.num_head = num_head;
  params.kv_num_head = k_num_head;
  params.timestep = timestep;
  params.seq_len = seq_len;
  params.max_seq_length = max_seq_len;
  params.inv_sqrt_dh = inv_sqrt_dh;
  params.rotary_emb_dims = rotary_emb_dims;

  if (out_shift) {
    DispatchFMHA<T>(dev_ctx,
                    x,
                    *(out_shift.get_ptr()),
                    *(out_smooth.get_ptr()),
                    params,
                    num_head,
                    dim_head,
                    out,
                    qkv_out_scale.get_ptr(),
                    out_scale,
                    quant_round_type,
                    quant_max_bound,
                    quant_min_bound);
  } else {
    DispatchFMHA<T>(dev_ctx,
                    x,
                    params,
                    num_head,
                    dim_head,
                    out,
                    qkv_out_scale.get_ptr(),
                    out_scale,
                    quant_round_type,
                    quant_max_bound,
                    quant_min_bound);
  }
}

template <typename T, typename Context>
void DispatchWithDtype(const Context &dev_ctx,
                       const DenseTensor &x,
                       const DenseTensor &cache_kv,
                       const paddle::optional<DenseTensor> &bias,
                       const paddle::optional<DenseTensor> &src_mask,
                       const paddle::optional<DenseTensor> &cum_offsets,
                       const paddle::optional<DenseTensor> &sequence_lengths,
                       const paddle::optional<DenseTensor> &rotary_tensor,
                       const paddle::optional<DenseTensor> &beam_cache_offset,
                       const paddle::optional<DenseTensor> &qkv_out_scale,
                       const paddle::optional<DenseTensor> &out_shift,
                       const paddle::optional<DenseTensor> &out_smooth,
                       int seq_len,
                       int rotary_emb_dims,
                       const bool use_neox_rotary_style,
                       const float out_scale,
                       const int quant_round_type,
                       const float quant_max_bound,
                       const float quant_min_bound,
                       DenseTensor *out,
                       DenseTensor *cache_kv_out,
                       DenseTensor *beam_cache_offset_out,
                       UnusedVersion) {}

#endif  // PADDLE_WITH_HIP

template <typename T, typename Context>
void MMHAKernel(const Context &dev_ctx,
                const DenseTensor &x,
                const DenseTensor &cache_kv,
                const paddle::optional<DenseTensor> &bias,
                const paddle::optional<DenseTensor> &src_mask,
                const paddle::optional<DenseTensor> &cum_offsets,
                const paddle::optional<DenseTensor> &sequence_lengths,
                const paddle::optional<DenseTensor> &rotary_tensor,
                const paddle::optional<DenseTensor> &beam_cache_offset,
                const paddle::optional<DenseTensor> &qkv_out_scale,
                const paddle::optional<DenseTensor> &out_shift,
                const paddle::optional<DenseTensor> &out_smooth,
                int seq_len,
                int rotary_emb_dims,
                const bool use_neox_rotary_style,
                const std::string &compute_dtype,
                const float out_scale,
                const int quant_round_type,
                const float quant_max_bound,
                const float quant_min_bound,
                DenseTensor *out,
                DenseTensor *cache_kv_out,
                DenseTensor *beam_cache_offset_out) {
#ifndef PADDLE_WITH_HIP
  if (x.dtype() == phi::DataType::INT32) {
    switch (str2int(compute_dtype.c_str())) {
      case str2int("fp16"):
        DispatchWithDtype<phi::dtype::float16, Context>(
            dev_ctx,
            x,
            cache_kv,
            bias,
            src_mask,
            cum_offsets,
            sequence_lengths,
            rotary_tensor,
            beam_cache_offset,
            qkv_out_scale,
            out_shift,
            out_smooth,
            seq_len,
            rotary_emb_dims,
            use_neox_rotary_style,
            out_scale,
            quant_round_type,
            quant_max_bound,
            quant_min_bound,
            out,
            cache_kv_out,
            beam_cache_offset_out,
            typename DispatchDtypeTrait<phi::dtype::float16>::FuncVersion{});
        break;
#if CUDA_VERSION >= 11000
      case str2int("bf16"):
        DispatchWithDtype<phi::dtype::bfloat16, Context>(
            dev_ctx,
            x,
            cache_kv,
            bias,
            src_mask,
            cum_offsets,
            sequence_lengths,
            rotary_tensor,
            beam_cache_offset,
            qkv_out_scale,
            out_shift,
            out_smooth,
            seq_len,
            rotary_emb_dims,
            use_neox_rotary_style,
            out_scale,
            quant_round_type,
            quant_max_bound,
            quant_min_bound,
            out,
            cache_kv_out,
            beam_cache_offset_out,
            typename DispatchDtypeTrait<phi::dtype::bfloat16>::FuncVersion{});
        break;
#endif
      case str2int("fp32"):
        DispatchWithDtype<float, Context>(
            dev_ctx,
            x,
            cache_kv,
            bias,
            src_mask,
            cum_offsets,
            sequence_lengths,
            rotary_tensor,
            beam_cache_offset,
            qkv_out_scale,
            out_shift,
            out_smooth,
            seq_len,
            rotary_emb_dims,
            use_neox_rotary_style,
            out_scale,
            quant_round_type,
            quant_max_bound,
            quant_min_bound,
            out,
            cache_kv_out,
            beam_cache_offset_out,
            typename DispatchDtypeTrait<float>::FuncVersion{});
        break;
      default:
        PADDLE_THROW(phi::errors::InvalidArgument(
            "In the case of quantization enabled with Input(x) INT32, "
            "Attr(compute_dtype) must be set in (bf16, fp16, fp32), "
            "but get compute_dtype (%s)",
            compute_dtype));
    }
  } else {
    DispatchWithDtype<T, Context>(
        dev_ctx,
        x,
        cache_kv,
        bias,
        src_mask,
        cum_offsets,
        sequence_lengths,
        rotary_tensor,
        beam_cache_offset,
        qkv_out_scale,
        out_shift,
        out_smooth,
        seq_len,
        rotary_emb_dims,
        use_neox_rotary_style,
        out_scale,
        quant_round_type,
        quant_max_bound,
        quant_min_bound,
        out,
        cache_kv_out,
        beam_cache_offset_out,
        typename DispatchDtypeTrait<T>::FuncVersion{});
  }
#endif  // PADDLE_WITH_HIP
}

}  // namespace fusion
}  // namespace phi

#if CUDA_VERSION >= 11000
PD_REGISTER_KERNEL(masked_multihead_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::MMHAKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   int32_t) {}
#else
PD_REGISTER_KERNEL(masked_multihead_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::MMHAKernel,
                   float,
                   phi::dtype::float16,
                   int32_t) {}
#endif
