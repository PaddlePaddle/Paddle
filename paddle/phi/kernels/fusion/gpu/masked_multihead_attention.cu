#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/fusion/gpu/masked_multihead_attention.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {
namespace fusion {

template <typename T, typename Context>
void MMHAKernel(const Context& dev_ctx,
                const DenseTensor& x,
                const DenseTensor& cache_kv,
                const paddle::optional<DenseTensor>& src_mask,
                const paddle::optional<DenseTensor>& cum_offsets,
                const paddle::optional<DenseTensor>& sequence_lengths,
                const paddle::optional<DenseTensor>& rotary_tensor,
                const paddle::optional<DenseTensor>& beam_cache_offset,
                const paddle::optional<DenseTensor>& qkv_out_scale,
                const paddle::optional<DenseTensor>& out_shift,
                const paddle::optional<DenseTensor>& out_smooth,
                int seq_len,
                int rotary_emb_dims,
                const bool use_neox_rotary_style,
                const float out_scale,
                const int quant_round_type,
                const float quant_max_bound,
                const float quant_min_bound,
                DenseTensor* out,
                DenseTensor* cache_kv_out,
                DenseTensor* beam_cache_offset_out) {
#ifndef PADDLE_WITH_HIP
  const auto& x_dims = x.dims();
  int bsz = x_dims[0];
  int cache_bsz = cache_kv.dims()[1];
  int num_head = cache_kv.dims()[2];
  int max_seq_len = cache_kv.dims()[3];
  int dim_head = cache_kv.dims()[4];
  int timestep = max_seq_len;
  float inv_sqrt_dh = 1. / sqrt(dim_head);

  Masked_multihead_attention_params<T> params;
  bool mask_broadcast_num_heads = true;

  if (src_mask) {
    if (src_mask->dims()[1] == 1) {
      mask_broadcast_num_heads = true;
    } else if (src_mask->dims()[1] == num_head) {
      mask_broadcast_num_heads = false;
    } else {
      PADDLE_THROW(errors::InvalidArgument(
          "Unknow dimension for attn_mask, the num_head(2nd) "
          "dimension is invalid, it should be 1 or num_head(%d), "
          "but got %d",
          num_head,
          src_mask->dims()[1]));
    }
    params.attn_mask = src_mask->data<T>();
    params.mask_length = src_mask->dims()[3];
    timestep = src_mask->dims()[3] - 1;
  }

  if (out_scale > 0) {
    dev_ctx.template Alloc<int8_t>(out);
  } else {
    dev_ctx.template Alloc<T>(out);
  }

  if (sequence_lengths) {
    params.sequence_lengths = sequence_lengths->data<int>();
  }

  if (cum_offsets) {
    params.cum_offsets = cum_offsets->data<int>();
  } else {
    params.cum_offsets = nullptr;
  }

  if (rotary_emb_dims > 0) {
    params.rotary_emb = rotary_tensor->data<float>();
  } else {
    params.rotary_emb = nullptr;
  }

  if (beam_cache_offset) {
    params.beam_cache_offset = beam_cache_offset->data<int>();
    params.beam_width = beam_cache_offset->dims()[1];
  }

  params.mask_broadcast_num_heads = mask_broadcast_num_heads;
  params.cache_kv = const_cast<T*>(cache_kv_out->data<T>());
  params.neox_rotary_style = use_neox_rotary_style;
  params.add_qkv_bias = false;
  params.batch_size = bsz;
  params.cache_batch_size = cache_bsz;
  params.num_head = num_head;
  params.timestep = timestep;
  params.seq_len = seq_len;
  params.max_seq_length = max_seq_len;
  params.inv_sqrt_dh = inv_sqrt_dh;
  params.rotary_emb_dims = rotary_emb_dims;

  if (out_shift) {
    DispatchFMHA<T>(dev_ctx,
                    x,
                    *(out_shift.get_ptr()),
                    *(out_smooth.get_ptr()),
                    params,
                    num_head,
                    dim_head,
                    out,
                    qkv_out_scale.get_ptr(),
                    out_scale,
                    quant_round_type,
                    quant_max_bound,
                    quant_min_bound);
  } else {
    DispatchFMHA<T>(dev_ctx,
                    x,
                    params,
                    num_head,
                    dim_head,
                    out,
                    qkv_out_scale.get_ptr(),
                    out_scale,
                    quant_round_type,
                    quant_max_bound,
                    quant_min_bound);
  }
#endif  // PADDLE_WITH_HIP
}

}  // namespace fusion
}  // namespace phi

#if CUDA_VERSION >= 11000
PD_REGISTER_KERNEL(masked_multihead_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::MMHAKernel,
                   float,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#else
PD_REGISTER_KERNEL(masked_multihead_attention,
                   GPU,
                   ALL_LAYOUT,
                   phi::fusion::MMHAKernel,
                   float,
                   phi::dtype::float16) {}
#endif
