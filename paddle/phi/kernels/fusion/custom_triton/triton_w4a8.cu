// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_fp16.h>
#include <vector>
#include "generated/w4a8/w4a8.h"
#include "paddle/extension.h"

std::map<std::vector<int>, int> map_problem_triton_w4a8;

std::vector<paddle::Tensor> TritonW4a8(const paddle::Tensor& x,
                                        const paddle::Tensor& qweight,
                                        bool bool_trans_w) {
  int m = x.shape()[0];
  int k = x.shape()[1];
  int n = qweight.shape()[1];
  if (bool_trans_w) {
    n = qweight.shape()[0];
  }
  
  std::cout << "TritonW4a8: m=" << m << ", k=" << k << ", n=" << n << std::endl;
  auto c_out = paddle::full({m, n}, 0, paddle::DataType::INT32, x.place());

  auto dev_x = x.data<int8_t>();
  //auto dev_weight = qweight.data<int8_t>();
  auto dev_weight = qweight.data<int32_t>();
  auto dev_c = c_out.data<int32_t>();

  int stride_bk = n;
  int stride_bn = 1;

  if (bool_trans_w) {
    stride_bk = 1;
    //stride_bn = k / 8;
    stride_bn = k / 4; 
  }

  std::vector<int> problem_size = {m, k, n};

  if (map_problem_triton_w4a8.count(problem_size)) {
    int algo_id = map_problem_triton_w4a8[problem_size];
    printf("TritonW4a8: %d\n", algo_id);
    auto status = w4a8_kernel(c_out.stream(),
                               (hipDeviceptr_t)(dev_x),
                               (hipDeviceptr_t)(dev_weight),
                               (hipDeviceptr_t)(dev_c),
                               m,
                               n,
                               k,
                               k,
                               1,
                               stride_bk,
                               stride_bn,
                               n,
                               1,
                               algo_id);
    assert(status == hipSuccess);
    return {c_out};
  }

  float min_time = 10000.f;
  int select_id = -1;
  constexpr int WARMUP = 5;
  constexpr int REPEAT = 10;

  for (int algo_id = 0; algo_id < w4a8_kernel_get_num_algos(); ++algo_id) {
    hipEvent_t beg[REPEAT];
    hipEvent_t end[REPEAT];
    float elapsed_times[REPEAT];

    auto status = hipSuccess;

    for (int ii = 0; ii < WARMUP + REPEAT; ii++) {
      int repeat_id = ii - WARMUP;

      if (repeat_id >= 0) {
        (hipEventCreate(beg + repeat_id));
        (hipEventCreate(end + repeat_id));
        (hipEventRecord(beg[repeat_id]));
      }

      auto flush_l2_cache = paddle::full(
          {10 * 1024 * 1024}, 0, paddle::DataType::INT32, x.place());
      // std::cout << &flush_l2_cache  << std::endl;

      hipMemset(dev_c, 0, sizeof(phi::dtype::float16) * m * n);
      status = w4a8_kernel(c_out.stream(),
                            (hipDeviceptr_t)(dev_x),
                            (hipDeviceptr_t)(dev_weight),
                            (hipDeviceptr_t)(dev_c),
                            m,
                            n,
                            k,
                            k,
                            1,
                            stride_bk,
                            stride_bn,
                            n,
                            1,
                            algo_id);
      // assert(status == hipSuccess);

      if (repeat_id >= 0) {
        (hipEventRecord(end[repeat_id]));
        (hipEventSynchronize(end[repeat_id]));
        (hipEventElapsedTime(
            elapsed_times + repeat_id, beg[repeat_id], end[repeat_id]));
      }
    }

    float avg_elapsed_time = 0.f;
    for (int ii = 0; ii < REPEAT; ++ii) {
      avg_elapsed_time += elapsed_times[ii];
    }

    if (avg_elapsed_time < min_time && status == hipSuccess) {
      min_time = avg_elapsed_time;
      select_id = algo_id;
    }
  }

  map_problem_triton_w4a8[problem_size] = select_id;
  std::cout << "select algo id: " << select_id << std::endl;

  return {c_out};
}

PD_BUILD_OP(triton_w4a8)
    .Inputs({"x", "qweight"})
    .Outputs({"out"})
    .SetKernelFn(PD_KERNEL(TritonW4a8))
    .Attrs({"bool_trans_w: bool"});

