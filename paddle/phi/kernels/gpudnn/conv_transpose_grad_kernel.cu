#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/conv_transpose_grad_kernel.h"

#include <algorithm>

#include "paddle/phi/backends/dynload/hipDNN.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cpu/conv_util.h"
#include "paddle/phi/kernels/funcs/batch_norm_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/padding.h"
#include "paddle/phi/kernels/funcs/slice.h"
#include "paddle/phi/kernels/transpose_kernel.h"

#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/conv_miopen_helper.h"
#include "paddle/fluid/platform/device/gpu/rocm/miopen_helper.h"
#else
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#include "paddle/fluid/platform/device/gpu/cuda/cudnn_helper.h"
#endif

namespace phi {

using GPUDNNDataLayout = paddle::platform::DataLayout;

template <typename T, typename Context>
void ConvTransposeGradRawGPUDNNKernel(const Context& ctx,
                                      const DenseTensor& x,
                                      const DenseTensor& filter,
                                      const DenseTensor& dout,
                                      const std::vector<int>& strides,
                                      const std::vector<int>& paddings,
                                      const std::string& padding_algorithm,
                                      int groups,
                                      const std::vector<int>& dilations,
                                      const std::string& data_format,
                                      DenseTensor* dx,
                                      DenseTensor* dfilter) {
  const T* filter_data = filter.data<T>();
  std::vector<int> paddings_ = paddings;
  std::vector<int> dilations_ =
      dilations;  // cudnn v5 does not support dilations
  const GPUDNNDataLayout data_layout =
      (data_format != "NHWC" ? GPUDNNDataLayout::kNCHW
                             : GPUDNNDataLayout::kNHWC);

  // if channel_last, transpose to channel_first
  DenseTensor x_transpose;
  DenseTensor dout_transpose;
  std::vector<int> x_vec = vectorize<int>(x.dims());
  std::vector<int> out_vec = vectorize<int>(dout.dims());
  if (data_layout == GPUDNNDataLayout::kNHWC) {
    if (strides.size() == 2U) {
      std::vector<int> axis = {0, 3, 1, 2};
      for (size_t i = 0; i < axis.size(); ++i) {
        x_vec[i] = x.dims()[axis[i]];
        out_vec[i] = dout.dims()[axis[i]];
      }
      x_transpose = Transpose<T, Context>(ctx, x, axis);
      dout_transpose = Transpose<T, Context>(ctx, dout, axis);
    } else if (strides.size() == 3U) {
      std::vector<int> axis = {0, 4, 1, 2, 3};
      for (size_t i = 0; i < axis.size(); ++i) {
        x_vec[i] = x.dims()[axis[i]];
        out_vec[i] = dout.dims()[axis[i]];
      }
      x_transpose = Transpose<T, Context>(ctx, x, axis);
      dout_transpose = Transpose<T, Context>(ctx, dout, axis);
    }
  } else {
    x_transpose = x;
    dout_transpose = dout;
  }

  // update padding and dilation
  auto x_dims = x_transpose.dims();
  auto filter_dims = filter.dims();
  DDim x_data_dims;
  x_data_dims = slice_ddim(x_dims, 2, x_dims.size());
  DDim filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings_, &dilations_, padding_algorithm, x_data_dims, strides, ksize);

  int data_dim = strides.size();  // 2d or 3d
  bool is_sys_pad = funcs::IsSymmetricPadding(paddings_, data_dim);

  std::vector<int> x_pad(x_dims.size() * 2, 0);
  DenseTensor transformed_dout;
  std::vector<int> padding_common(data_dim, 0);
  if (!is_sys_pad) {
    std::vector<int> padding_diff(data_dim);
    std::vector<int> new_dout_shape_vec(data_dim + 2);
    new_dout_shape_vec[0] = dout_transpose.dims()[0];
    new_dout_shape_vec[1] = dout_transpose.dims()[1];

    for (size_t i = 0; i < data_dim; ++i) {
      padding_diff[i] = std::abs(paddings_[2 * i] - paddings_[2 * i + 1]);
      padding_common[i] = std::min(paddings_[2 * i], paddings_[2 * i + 1]);
      new_dout_shape_vec[i + 2] =
          dout_transpose.dims()[i + 2] + padding_diff[i];
      x_pad[2 * i + 4] = paddings_[2 * i] - padding_common[i];
      x_pad[2 * i + 4 + 1] = paddings_[2 * i + 1] - padding_common[i];
    }

    transformed_dout.Resize(make_ddim(new_dout_shape_vec));
    ctx.template Alloc<T>(&transformed_dout);

    const int rank = x_transpose.dims().size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(
            ctx, x_pad, dout_transpose, pad_value, &transformed_dout);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(
            ctx, x_pad, dout_transpose, pad_value, &transformed_dout);
      } break;
      default:
        PADDLE_THROW(errors::InvalidArgument(
            "Op(ConvTranspose) only supports 4-D or 5-D x DenseTensor."));
    }
  } else {
    transformed_dout = dout_transpose;
    if (paddings_.size() == data_dim) {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings_[i];
      }
    } else {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings_[2 * i];
      }
    }
  }

  const T* x_data = x_transpose.data<T>();
  const T* dout_data = transformed_dout.data<T>();
  out_vec = vectorize<int>(transformed_dout.dims());

  // ------------------- cudnn descriptors ---------------------
  GPUDNNDataLayout layout;

  if (strides.size() == 2U) {
    layout = GPUDNNDataLayout::kNCHW;
  } else {
    layout = GPUDNNDataLayout::kNCDHW;
  }

  int iwo_groups = groups;
  int c_groups = 1;
#if defined(PADDLE_WITH_HIP) || CUDNN_VERSION_MIN(7, 0, 1)
  iwo_groups = 1;
  c_groups = groups;
  groups = 1;
#endif

  auto dtype = paddle::platform::CudnnDataType<T>::type;

  paddle::operators::ConvArgs args1{&transformed_dout,
                                    &filter,
                                    &x_transpose,
                                    strides,
                                    padding_common,
                                    dilations_,
                                    dtype,
                                    groups,
                                    layout};
  paddle::operators::ConvArgs args2{&transformed_dout,
                                    &filter,
                                    &x_transpose,
                                    strides,
                                    padding_common,
                                    dilations_,
                                    dtype,
                                    groups,
                                    layout};

#ifdef PADDLE_WITH_HIP
  paddle::operators::SearchResult<miopenConvFwdAlgorithm_t> fwd_result;
  paddle::operators::SearchResult<miopenConvBwdWeightsAlgorithm_t>
      filter_result;
#else
  paddle::operators::SearchResult<hipdnnConvolutionFwdAlgo_t> fwd_result;
  paddle::operators::SearchResult<hipdnnConvolutionBwdFilterAlgo_t>
      filter_result;
#endif

  auto layout_tensor = paddle::platform::GetCudnnTensorFormat(layout);
  size_t workspace_size = 0;
  auto handle = ctx.cudnn_handle();
  bool deterministic = FLAGS_cudnn_deterministic;
  T* dx_data = nullptr;
  T* dfilter_data = nullptr;

  if (dx) {
    dx_data = ctx.template Alloc<T>(dx);
    args1.handle = handle;
    args1.idesc.set(transformed_dout, iwo_groups);
    args1.wdesc.set(filter, layout_tensor, iwo_groups);
    args1.odesc.set(x_transpose, iwo_groups);
    args1.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations_,
                    paddle::platform::AllowTF32Cudnn(),
                    c_groups);
#ifdef PADDLE_WITH_HIP
    using search1 =
        paddle::operators::SearchAlgorithm<miopenConvFwdAlgorithm_t>;
    workspace_size = std::max(workspace_size, search1::GetWorkspaceSize(args1));
    fwd_result.algo =
        search1::Find<T>(args1, false, deterministic, workspace_size, ctx);
#else
    using search1 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
    fwd_result = search1::Find<T>(ctx, args1, false, deterministic, false);
    workspace_size = std::max(
        workspace_size, search1::GetWorkspaceSize(args1, fwd_result.algo));
#endif
  }

  if (dfilter) {
    dfilter_data = ctx.template Alloc<T>(dfilter);
    args2.handle = handle;
    args2.idesc.set(transformed_dout, iwo_groups);
    args2.wdesc.set(*dfilter, layout_tensor, iwo_groups);
    args2.odesc.set(x_transpose, iwo_groups);
    args2.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations_,
                    paddle::platform::AllowTF32Cudnn(),
                    c_groups);
#ifdef PADDLE_WITH_HIP
    using search2 =
        paddle::operators::SearchAlgorithm<miopenConvBwdWeightsAlgorithm_t>;
    workspace_size = std::max(workspace_size, search2::GetWorkspaceSize(args2));
    filter_result.algo =
        search2::Find<T>(args2, false, deterministic, workspace_size, ctx);
#else
    using search2 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
    filter_result = search2::Find<T>(ctx, args2, false, deterministic, false);
    workspace_size = std::max(
        workspace_size, search2::GetWorkspaceSize(args2, filter_result.algo));
#endif
  }

  // ------------------- cudnn conv backward data ---------------------
  // FIxME(typhoonzero): template type T may not be the same as cudnn call.
  int x_offset = x.numel() / x.dims()[0] / groups;
  int dout_offset =
      transformed_dout.numel() / transformed_dout.dims()[0] / groups;
  int filter_offset = filter.numel() / groups;
  paddle::operators::ScalingParamType<T> alpha = 1.0f;
  paddle::operators::ScalingParamType<T> beta = 0.0f;
  auto workspace_handle = ctx.cudnn_workspace_handle();
  if (dx) {
    // Because beta is zero, it is unnecessary to reset dx.
    for (int g = 0; g < groups; g++) {
#ifdef PADDLE_WITH_HIP
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_GPU_SUCCESS(
            dynload::miopenConvolutionForward(handle,
                                              &alpha,
                                              args1.idesc.desc(),
                                              dout_data + dout_offset * g,
                                              args1.wdesc.desc(),
                                              filter_data + filter_offset * g,
                                              args1.cdesc.desc(),
                                              fwd_result.algo,
                                              &beta,
                                              args1.odesc.desc(),
                                              dx_data + x_offset * g,
                                              cudnn_workspace,
                                              workspace_size));
      };
#else   // PADDLE_WITH_HIP
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_GPU_SUCCESS(
            dynload::hipdnnConvolutionForward(handle,
                                             &alpha,
                                             args1.idesc.desc(),
                                             dout_data + dout_offset * g,
                                             args1.wdesc.desc(),
                                             filter_data + filter_offset * g,
                                             args1.cdesc.desc(),
                                             fwd_result.algo,
                                             cudnn_workspace,
                                             workspace_size,
                                             &beta,
                                             args1.odesc.desc(),
                                             dx_data + x_offset * g));
      };
#endif  // PADDLE_WITH_HIP
      workspace_handle.RunFunc(cudnn_func, workspace_size);
    }

    if (data_layout == GPUDNNDataLayout::kNHWC) {
      DenseTensor dx_transpose;
      DenseTensor dx_nchw;
      dx_nchw.ShareDataWith(*dx);
      dx_nchw.Resize(make_ddim(x_vec));
      if (strides.size() == 2U) {
        std::vector<int> axis = {0, 2, 3, 1};
        dx_transpose = Transpose<T, Context>(ctx, dx_nchw, axis);
        *dx = dx_transpose;
      } else if (strides.size() == 3U) {
        std::vector<int> axis = {0, 2, 3, 4, 1};
        dx_transpose = Transpose<T, Context>(ctx, dx_nchw, axis);
        *dx = dx_transpose;
      }
    }
  }

  // ------------------- cudnn conv backward filter ---------------------
  if (dfilter) {
    // Because beta is zero, it is unnecessary to reset dfilter.
    // Gradient with respect to the filter
    for (int g = 0; g < groups; g++) {
#ifdef PADDLE_WITH_HIP
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenConvolutionBackwardWeights(
            handle,
            &alpha,
            args2.odesc.desc(),
            x_data + x_offset * g,
            args2.idesc.desc(),
            dout_data + dout_offset * g,
            args2.cdesc.desc(),
            filter_result.algo,
            &beta,
            args2.wdesc.desc(),
            dfilter_data + filter_offset * g,
            cudnn_workspace,
            workspace_size));
      };
#else   // PADDLE_WITH_HIP
      auto cudnn_func = [&](void* cudnn_workspace) {
        PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnConvolutionBackwardFilter(
            handle,
            &alpha,
            args2.idesc.desc(),
            dout_data + dout_offset * g,
            args2.odesc.desc(),
            x_data + x_offset * g,
            args2.cdesc.desc(),
            filter_result.algo,
            cudnn_workspace,
            workspace_size,
            &beta,
            args2.wdesc.desc(),
            dfilter_data + filter_offset * g));
      };
#endif  // PADDLE_WITH_HIP
      workspace_handle.RunFunc(cudnn_func, workspace_size);
    }
  }
}

template <typename T, typename Context>
void Conv2dTransposeGradGPUDNNKernel(const Context& ctx,
                                     const DenseTensor& x,
                                     const DenseTensor& filter,
                                     const DenseTensor& dout,
                                     const std::vector<int>& strides,
                                     const std::vector<int>& paddings_,
                                     const std::vector<int>& output_padding,
                                     const IntArray& output_size,
                                     const std::string& padding_algorithm,
                                     int groups,
                                     const std::vector<int>& dilations_,
                                     const std::string& data_format,
                                     DenseTensor* dx,
                                     DenseTensor* dfilter) {
  ConvTransposeGradRawGPUDNNKernel<T, Context>(ctx,
                                               x,
                                               filter,
                                               dout,
                                               strides,
                                               paddings_,
                                               padding_algorithm,
                                               groups,
                                               dilations_,
                                               data_format,
                                               dx,
                                               dfilter);
}

/*
 * Inputs:  I, filter, dout, ddI, ddfilter
 * Outputs: ddout, dfilter, dI
 * ddo = conv_bp_data(filter, ddI) + conv_bp_data(ddfilter, I)
 * dfilter = conv_bp_filter(dout, ddI)
 * dI = conv(dout, ddfilter)
 */
template <typename T, typename Context>
void Conv2dTransposeDoubleGradGPUDNNKernel(
    const Context& ctx,
    const DenseTensor& x,
    const DenseTensor& filter,
    const DenseTensor& dout,
    const DenseTensor& ddx,
    const DenseTensor& ddfilter,
    const std::vector<int>& strides,
    const std::vector<int>& paddings,
    const std::vector<int>& output_padding,
    const IntArray& output_size,
    const std::string& padding_algorithm,
    int groups,
    const std::vector<int>& dilations,
    const std::string& data_format,
    DenseTensor* dx,
    DenseTensor* dfilter,
    DenseTensor* ddout) {
  if (dx) {
    ctx.template Alloc<T>(dx);
  }
  if (dfilter) {
    ctx.template Alloc<T>(dfilter);
  }
  if (ddout) {
    ctx.template Alloc<T>(ddout);
    funcs::SetConstant<Context, T> set_zero;
    set_zero(ctx, ddout, static_cast<T>(0));
  }

  const T* filter_ = filter.data<T>();
  const T* dout_ = dout.data<T>();
  const T* ddx_ = nullptr;
  const T* ddfilter_ = nullptr;
  T* dx_ = nullptr;
  T* dfilter_ = nullptr;
  T* ddout_ = nullptr;
  T* transformed_dx_ = nullptr;

  std::vector<int> paddings_ = paddings;
  std::vector<int> dilations_ = dilations;

  bool deterministic = FLAGS_cudnn_deterministic;
  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

  // transform DenseTensors to channel first-----------
  DenseTensor transformed_x_channel(x.type());
  DenseTensor transformed_dout_channel(dout.type());
  DenseTensor transformed_ddx_channel(x.type());

  DenseTensor transformed_dx_channel(x.type());
  DenseTensor transformed_ddout_channel(dout.type());

  if (channel_last) {
    ResizeToChannelFirst<Context, T>(ctx, &x, &transformed_x_channel);
    TransToChannelFirst<Context, T>(ctx, &x, &transformed_x_channel);

    ResizeToChannelFirst<Context, T>(ctx, &dout, &transformed_dout_channel);
    TransToChannelFirst<Context, T>(ctx, &dout, &transformed_dout_channel);

    ResizeToChannelFirst<Context, T>(ctx, &ddx, &transformed_ddx_channel);
    TransToChannelFirst<Context, T>(ctx, &ddx, &transformed_ddx_channel);

    if (dx) {
      ResizeToChannelFirst<Context, T>(ctx, dx, &transformed_dx_channel);
      ctx.template Alloc<T>(&transformed_dx_channel);
    }
    if (ddout) {
      ResizeToChannelFirst<Context, T>(ctx, ddout, &transformed_ddout_channel);
    }
  } else {
    transformed_x_channel = x;
    transformed_dout_channel = dout;
    transformed_ddx_channel = ddx;

    if (dx) {
      transformed_dx_channel = *dx;
    }
  }
  std::vector<int> out_vec = vectorize<int>(transformed_dout_channel.dims());

  auto x_dims = transformed_x_channel.dims();
  auto filter_dims = filter.dims();
  DDim x_data_dims = slice_ddim(x_dims, 2, x_dims.size());
  DDim filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings_, &dilations_, padding_algorithm, x_data_dims, strides, ksize);

  int data_dim = strides.size();  // 2d or 3d
  bool is_sys_pad = funcs::IsSymmetricPadding(paddings_, data_dim);
  DenseTensor transformed_x(x.type());
  DenseTensor transformed_ddx(x.type());

  DenseTensor transformed_dout(dout.type());

  std::vector<int> padding_common(data_dim, 0);
  std::vector<int> input_pad(x.dims().size() * 2, 0);

  if (!is_sys_pad) {
    // get pad
    std::vector<int> padding_diff(data_dim);
    std::vector<int> new_input_shape_vec(data_dim + 2);
    std::vector<int> new_output_grad_shape_vec(data_dim + 2);

    new_input_shape_vec[0] = transformed_x_channel.dims()[0];
    new_input_shape_vec[1] = transformed_x_channel.dims()[1];

    new_output_grad_shape_vec[0] = transformed_dout_channel.dims()[0];
    new_output_grad_shape_vec[1] = transformed_dout_channel.dims()[1];

    for (size_t i = 0; i < data_dim; ++i) {
      padding_diff[i] = std::abs(paddings_[2 * i] - paddings_[2 * i + 1]);
      padding_common[i] = std::min(paddings_[2 * i], paddings_[2 * i + 1]);
      new_input_shape_vec[i + 2] =
          transformed_x_channel.dims()[i + 2] + padding_diff[i];

      new_output_grad_shape_vec[i + 2] =
          transformed_dout_channel.dims()[i + 2] + padding_diff[i];

      input_pad[2 * i + 4] = paddings_[2 * i] - padding_common[i];
      input_pad[2 * i + 4 + 1] = paddings_[2 * i + 1] - padding_common[i];
    }
    DDim new_input_shape(make_ddim(new_input_shape_vec));
    transformed_x.Resize(new_input_shape);
    transformed_ddx.Resize(new_input_shape);
    transformed_dout.Resize(make_ddim(new_output_grad_shape_vec));

    ctx.template Alloc<T>(&transformed_x);
    ctx.template Alloc<T>(&transformed_ddx);
    ctx.template Alloc<T>(&transformed_dout);

    // pad for input
    const int rank = x.dims().size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(
            ctx, input_pad, transformed_x_channel, pad_value, &transformed_x);
        funcs::PadFunction<Context, T, 4>(ctx,
                                          input_pad,
                                          transformed_dout_channel,
                                          pad_value,
                                          &transformed_dout);
        funcs::PadFunction<Context, T, 4>(ctx,
                                          input_pad,
                                          transformed_ddx_channel,
                                          pad_value,
                                          &transformed_ddx);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(
            ctx, input_pad, transformed_x_channel, pad_value, &transformed_x);
        funcs::PadFunction<Context, T, 5>(ctx,
                                          input_pad,
                                          transformed_ddx_channel,
                                          pad_value,
                                          &transformed_ddx);
      } break;
      default:
        PADDLE_THROW(errors::InvalidArgument(
            "ConvOp only support tensors with 4 or 5 dimensions."));
    }
  } else {
    transformed_x = transformed_x_channel;
    transformed_dout = transformed_dout_channel;
    transformed_ddx = transformed_ddx_channel;

    if (paddings_.size() == data_dim) {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings_[i];
      }
    } else {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings_[2 * i];
      }
    }
  }

  std::vector<int64_t> starts(data_dim, 0);
  std::vector<int64_t> ends(data_dim, 0);
  std::vector<int64_t> axes(data_dim, 0);
  for (size_t i = 0; i < data_dim; ++i) {
    starts[i] = input_pad[2 * i + 4] * (strides[i] + 1);
    ends[i] = starts[i] + out_vec[i + 2];
    axes[i] = i + 2;
  }

  std::vector<int> transformed_out_vec = out_vec;
  for (size_t i = 0; i < data_dim; ++i) {
    transformed_out_vec[i + 2] =
        out_vec[i + 2] +
        (input_pad[2 * i + 4] + input_pad[2 * i + 5]) * strides[i] -
        2 * padding_common[i] + paddings_[2 * i] + paddings_[2 * i + 1];
  }

  if (!is_sys_pad) {
    transformed_ddout_channel.Resize(make_ddim(transformed_out_vec));
    ctx.template Alloc<T>(&transformed_ddout_channel);
  } else {
    ctx.template Alloc<T>(ddout);
    transformed_ddout_channel = *ddout;
    transformed_ddout_channel.Resize(make_ddim(transformed_out_vec));
  }

  const T* x_ = transformed_x.data<T>();

  int iwo_group = groups;
  int c_group = 1;
#if defined(PADDLE_WITH_HIP) || CUDNN_VERSION_MIN(7, 0, 1)
  iwo_group = 1;
  c_group = groups;
  groups = 1;
#endif
  auto dtype = paddle::platform::CudnnDataType<T>::type;

  auto handle = ctx.cudnn_handle();
  auto layout = paddle::platform::GetCudnnTensorFormat(GPUDNNDataLayout::kNCHW);

  paddle::operators::ConvArgs args1{&transformed_ddout_channel,
                                    &filter,
                                    &transformed_ddx,
                                    strides,
                                    padding_common,
                                    dilations_,
                                    dtype,
                                    groups,
                                    GPUDNNDataLayout::kNCHW};
  paddle::operators::ConvArgs args2{&transformed_ddout_channel,
                                    &ddfilter,
                                    &transformed_x,
                                    strides,
                                    padding_common,
                                    dilations_,
                                    dtype,
                                    groups,
                                    GPUDNNDataLayout::kNCHW};

  paddle::operators::ConvArgs args3{&transformed_dout,
                                    dfilter,
                                    &transformed_ddx_channel,
                                    strides,
                                    padding_common,
                                    dilations_,
                                    dtype,
                                    groups,
                                    GPUDNNDataLayout::kNCHW};
  paddle::operators::ConvArgs args4{&transformed_dout,
                                    &ddfilter,
                                    &transformed_dx_channel,
                                    strides,
                                    padding_common,
                                    dilations_,
                                    dtype,
                                    groups,
                                    GPUDNNDataLayout::kNCHW};
#ifdef PADDLE_WITH_HIP
  paddle::operators::SearchResult<miopenConvBwdDataAlgorithm_t> bwd_result1;
  paddle::operators::SearchResult<miopenConvBwdDataAlgorithm_t> bwd_result2;
  paddle::operators::SearchResult<miopenConvBwdWeightsAlgorithm_t>
      filter_result;
  paddle::operators::SearchResult<miopenConvFwdAlgorithm_t> fwd_result;
#else
  paddle::operators::SearchResult<hipdnnConvolutionBwdDataAlgo_t> bwd_result1;
  paddle::operators::SearchResult<hipdnnConvolutionBwdDataAlgo_t> bwd_result2;
  paddle::operators::SearchResult<hipdnnConvolutionBwdFilterAlgo_t>
      filter_result;
  paddle::operators::SearchResult<hipdnnConvolutionFwdAlgo_t> fwd_result;
#endif

  // ddo = conv(ddI, filter) + conv(I, ddfilter)
  size_t workspace_size = 0;

  T* transformed_ddout_channel_ = nullptr;

  if (ddout) {
    ddout_ = ddout->data<T>();
    transformed_ddout_channel_ = transformed_ddout_channel.data<T>();

    args1.handle = handle;
    args1.idesc.set(transformed_ddout_channel, iwo_group);
    args1.wdesc.set(filter, layout, iwo_group);
    args1.odesc.set(transformed_ddx, iwo_group);
    args1.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations_,
                    paddle::platform::AllowTF32Cudnn(),
                    c_group);
#ifdef PADDLE_WITH_HIP
    using search1 =
        paddle::operators::SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
    workspace_size = search1::GetWorkspaceSize(args1);
    bwd_result1.algo =
        search1::Find<T>(args1, false, deterministic, workspace_size, ctx);
#else
    using search1 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
    bwd_result1 = search1::Find<T>(ctx, args1, false, deterministic, false);
    workspace_size = search1::GetWorkspaceSize(args1, bwd_result1.algo);
#endif

    ddfilter_ = ddfilter.data<T>();
    args2.handle = handle;
    args2.idesc.set(transformed_ddout_channel, iwo_group);
    args2.wdesc.set(ddfilter, layout, iwo_group);
    args2.odesc.set(transformed_x, iwo_group);
    args2.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations_,
                    paddle::platform::AllowTF32Cudnn(),
                    c_group);
#ifdef PADDLE_WITH_HIP
    using search2 =
        paddle::operators::SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
    workspace_size = std::max(workspace_size, search2::GetWorkspaceSize(args2));
    bwd_result2.algo =
        search2::Find<T>(args2, false, deterministic, workspace_size, ctx);
#else
    using search2 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
    bwd_result2 = search2::Find<T>(ctx, args2, false, deterministic, false);
    workspace_size = std::max(
        workspace_size, search2::GetWorkspaceSize(args2, bwd_result2.algo));
#endif
  }

  if (dfilter) {
    dfilter_ = dfilter->data<T>();
    args3.handle = handle;
    args3.idesc.set(transformed_dout, iwo_group);
    args3.wdesc.set(*dfilter, layout, iwo_group);
    args3.odesc.set(transformed_ddx_channel, iwo_group);
    args3.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations_,
                    paddle::platform::AllowTF32Cudnn(),
                    c_group);
#ifdef PADDLE_WITH_HIP
    using search3 =
        paddle::operators::SearchAlgorithm<miopenConvBwdWeightsAlgorithm_t>;
    workspace_size = std::max(workspace_size, search3::GetWorkspaceSize(args3));
    filter_result.algo =
        search3::Find<T>(args3, false, deterministic, workspace_size, ctx);
#else
    using search3 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
    filter_result = search3::Find<T>(ctx, args3, false, deterministic, false);
    workspace_size = std::max(
        workspace_size, search3::GetWorkspaceSize(args3, filter_result.algo));
#endif
  }

  if (dx) {
    transformed_dx_ = transformed_dx_channel.data<T>();

    args4.handle = handle;
    args4.idesc.set(transformed_dout, iwo_group);
    args4.wdesc.set(ddfilter, layout, iwo_group);
    args4.odesc.set(transformed_dx_channel, iwo_group);
    args4.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations_,
                    paddle::platform::AllowTF32Cudnn(),
                    c_group);
#ifdef PADDLE_WITH_HIP
    using search4 =
        paddle::operators::SearchAlgorithm<miopenConvFwdAlgorithm_t>;
    workspace_size = std::max(workspace_size, search4::GetWorkspaceSize(args4));
    fwd_result.algo =
        search4::Find<T>(args4, false, deterministic, workspace_size, ctx);
#else
    using search4 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
    fwd_result = search4::Find<T>(ctx, args4, false, deterministic, false);
    workspace_size = std::max(
        workspace_size, search4::GetWorkspaceSize(args4, fwd_result.algo));
#endif
  }

  int i_n, i_c, i_d, i_h, i_w;
  paddle::operators::GetNCDHW(transformed_x.dims(),
                              GPUDNNDataLayout::kNCHW,
                              &i_n,
                              &i_c,
                              &i_d,
                              &i_h,
                              &i_w);

  int o_n, o_c, o_d, o_h, o_w;
  paddle::operators::GetNCDHW(transformed_dout.dims(),
                              GPUDNNDataLayout::kNCHW,
                              &o_n,
                              &o_c,
                              &o_d,
                              &o_h,
                              &o_w);

  int group_offset_in =
      transformed_x.numel() / transformed_x.dims()[0] / groups;
  int group_offset_out =
      transformed_dout.numel() / transformed_dout.dims()[0] / groups;
  int group_offset_filter = filter.numel() / groups;

  paddle::operators::ScalingParamType<T> alpha = 1.0f;
  paddle::operators::ScalingParamType<T> beta = 0.0f;

  auto wkspace_handle = ctx.cudnn_workspace_handle();

  if (ddout) {
    ddx_ = transformed_ddx.data<T>();
    for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenConvolutionBackwardData(
                handle,
                &alpha,
                args1.odesc.desc(),
                ddx_ + i * group_offset_in,
                args1.wdesc.desc(),
                filter_ + i * group_offset_filter,
                args1.cdesc.desc(),
                bwd_result1.algo,
                &beta,
                args1.idesc.desc(),
                transformed_ddout_channel_ + i * group_offset_out,
                workspace_ptr,
                workspace_size));
          },
          workspace_size);
#else   // PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnConvolutionBackwardData(
                handle,
                &alpha,
                args1.wdesc.desc(),
                filter_ + i * group_offset_filter,
                args1.odesc.desc(),
                ddx_ + i * group_offset_in,
                args1.cdesc.desc(),
                bwd_result1.algo,
                workspace_ptr,
                workspace_size,
                &beta,
                args1.idesc.desc(),
                transformed_ddout_channel_ + i * group_offset_out));
          },
          workspace_size);
#endif  // PADDLE_WITH_HIP
    }

    for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
      // MIOPEN ONLY support beta to be 0.0f
      DenseTensor conv_x_ddfilter(dout.type());
      conv_x_ddfilter.Resize(transformed_ddout_channel.dims());
      T* conv_x_ddfilter_data = ctx.template Alloc<T>(&conv_x_ddfilter);
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenConvolutionBackwardData(
                handle,
                &alpha,
                args2.odesc.desc(),
                x_ + i * group_offset_in,
                args2.wdesc.desc(),
                ddfilter_ + i * group_offset_filter,
                args2.cdesc.desc(),
                bwd_result2.algo,
                &beta,
                args2.idesc.desc(),
                conv_x_ddfilter_data + i * group_offset_out,
                workspace_ptr,
                workspace_size));
          },
          workspace_size);
      PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenOpTensor(
          handle,
          miopenTensorOpAdd,
          &alpha,
          args2.idesc.desc(),
          transformed_ddout_channel_ + i * group_offset_out,
          &alpha,
          args2.idesc.desc(),
          conv_x_ddfilter_data + i * group_offset_out,
          &beta,
          args2.idesc.desc(),
          transformed_ddout_channel_ + i * group_offset_out));
#else   // PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnConvolutionBackwardData(
                handle,
                &alpha,
                args2.wdesc.desc(),
                ddfilter_ + i * group_offset_filter,
                args2.odesc.desc(),
                x_ + i * group_offset_in,
                args2.cdesc.desc(),
                bwd_result2.algo,
                workspace_ptr,
                workspace_size,
                &alpha,
                args2.idesc.desc(),
                transformed_ddout_channel_ + i * group_offset_out));
          },
          workspace_size);
#endif  // PADDLE_WITH_HIP
    }

    if ((!is_sys_pad) && (!channel_last)) {
      if (strides.size() == 2U) {
        funcs::Slice<Context, T, 4>(
            ctx, &transformed_ddout_channel, ddout, starts, ends, axes);
      } else if (!is_sys_pad && strides.size() == 3U) {
        funcs::Slice<Context, T, 5>(
            ctx, &transformed_ddout_channel, ddout, starts, ends, axes);
      }
    } else if ((!is_sys_pad) && (channel_last)) {
      if (strides.size() == 2U) {
        funcs::Slice<Context, T, 4>(ctx,
                                    &transformed_ddout_channel,
                                    &transformed_ddout_channel,
                                    starts,
                                    ends,
                                    axes);
      } else if (!is_sys_pad && strides.size() == 3U) {
        funcs::Slice<Context, T, 5>(ctx,
                                    &transformed_ddout_channel,
                                    &transformed_ddout_channel,
                                    starts,
                                    ends,
                                    axes);
      }

      TransToChannelLast<Context, T>(ctx, &transformed_ddout_channel, ddout);
    }
  }

  T* transformed_dout_channel_ = transformed_dout.data<T>();
  if (dfilter) {
    ddx_ = transformed_ddx_channel.data<T>();
    for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                dynload::miopenConvolutionBackwardWeights(
                    handle,
                    &alpha,
                    args3.odesc.desc(),
                    ddx_ + i * group_offset_in,
                    args3.idesc.desc(),
                    transformed_dout_channel_ + i * group_offset_out,
                    args3.cdesc.desc(),
                    filter_result.algo,
                    &beta,
                    args3.wdesc.desc(),
                    dfilter_ + i * group_offset_filter,
                    workspace_ptr,
                    workspace_size));
          },
          workspace_size);
#else   // PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnConvolutionBackwardFilter(
                handle,
                &alpha,
                args3.idesc.desc(),
                transformed_dout_channel_ + i * group_offset_out,
                args3.odesc.desc(),
                ddx_ + i * group_offset_in,
                args3.cdesc.desc(),
                filter_result.algo,
                workspace_ptr,
                workspace_size,
                &beta,
                args3.wdesc.desc(),
                dfilter_ + i * group_offset_filter));
          },
          workspace_size);
#endif  // PADDLE_WITH_HIP
    }
  }

  if (dx) {
    ddfilter_ = ddfilter.data<T>();
    for (int i = 0; i < groups; i++) {
#ifdef PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenConvolutionForward(
                handle,
                &alpha,
                args4.idesc.desc(),
                transformed_dout_channel_ + i * group_offset_out,
                args4.wdesc.desc(),
                ddfilter_ + i * group_offset_filter,
                args4.cdesc.desc(),
                fwd_result.algo,
                &beta,
                args4.odesc.desc(),
                transformed_dx_ + i * group_offset_in,
                workspace_ptr,
                workspace_size));
          },
          workspace_size);
#else   // PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnConvolutionForward(
                handle,
                &alpha,
                args4.idesc.desc(),
                transformed_dout_channel_ + i * group_offset_out,
                args4.wdesc.desc(),
                ddfilter_ + i * group_offset_filter,
                args4.cdesc.desc(),
                fwd_result.algo,
                workspace_ptr,
                workspace_size,
                &beta,
                args4.odesc.desc(),
                transformed_dx_ + i * group_offset_in));
          },
          workspace_size);
#endif  // PADDLE_WITH_HIP
    }
    if (channel_last) {
      TransToChannelLast<Context, T>(ctx, &transformed_dx_channel, dx);
    }
  }
}

template <typename T, typename Context>
void Conv3dTransposeGradGPUDNNKernel(const Context& ctx,
                                     const DenseTensor& x,
                                     const DenseTensor& filter,
                                     const DenseTensor& dout,
                                     const std::vector<int>& strides,
                                     const std::vector<int>& paddings_,
                                     const std::vector<int>& output_padding,
                                     const std::vector<int>& output_size,
                                     const std::string& padding_algorithm,
                                     int groups,
                                     const std::vector<int>& dilations_,
                                     const std::string& data_format,
                                     DenseTensor* dx,
                                     DenseTensor* dfilter) {
  ConvTransposeGradRawGPUDNNKernel<T, Context>(ctx,
                                               x,
                                               filter,
                                               dout,
                                               strides,
                                               paddings_,
                                               padding_algorithm,
                                               groups,
                                               dilations_,
                                               data_format,
                                               dx,
                                               dfilter);
}

}  // namespace phi

using float16 = phi::dtype::float16;

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(conv2d_transpose_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv2dTransposeGradGPUDNNKernel,
                   float,
                   float16) {}
PD_REGISTER_KERNEL(conv2d_transpose_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv2dTransposeDoubleGradGPUDNNKernel,
                   float,
                   float16) {}
PD_REGISTER_KERNEL(conv3d_transpose_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3dTransposeGradGPUDNNKernel,
                   float,
                   float16) {}
#else
PD_REGISTER_KERNEL(conv2d_transpose_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv2dTransposeGradGPUDNNKernel,
                   float,
                   double,
                   float16) {}
PD_REGISTER_KERNEL(conv2d_transpose_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv2dTransposeDoubleGradGPUDNNKernel,
                   float,
                   double,
                   float16) {}
PD_REGISTER_KERNEL(conv3d_transpose_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3dTransposeGradGPUDNNKernel,
                   float,
                   double,
                   float16) {}
#endif
