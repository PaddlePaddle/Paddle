#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/conv_grad_grad_kernel.h"

#include "paddle/fluid/framework/eigen.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/conv_miopen_helper.h"
#else
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#endif

#include "paddle/fluid/platform/cudnn_workspace_helper.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/profiler.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/cpu/conv_util.h"
#include "paddle/phi/kernels/funcs/batch_norm_utils.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/padding.h"
#include "paddle/phi/kernels/impl/conv_cudnn_impl.h"

namespace phi {

template <typename T, typename Context>
void ConvCudnnGradGradKernel(
    const Context& ctx,
    const DenseTensor& input,
    const DenseTensor& filter,
    const DenseTensor& out_grad,
    const paddle::optional<DenseTensor>& input_grad_grad,
    const paddle::optional<DenseTensor>& filter_grad_grad,
    const std::vector<int>& strides,
    const std::vector<int>& paddings_t,
    const std::string& padding_algorithm,
    int groups,
    const std::vector<int>& dilations_t,
    const std::string& data_format,
    bool use_addto,
    int workspace_size_MB,
    bool exhaustive_search_t,
    DenseTensor* input_grad,
    DenseTensor* filter_grad,
    DenseTensor* out_grad_grad) {
  auto X = &input;
  auto W = &filter;
  auto dO = &out_grad;
  auto ddX = input_grad_grad.get_ptr();
  auto ddW = filter_grad_grad.get_ptr();

  auto ddO = out_grad_grad;
  auto dW = filter_grad;
  auto dX = input_grad;
  if (ddO) {
    ctx.template Alloc<T>(ddO);
    phi::funcs::SetConstant<Context, T> set_zero;
    set_zero(ctx, ddO, static_cast<T>(0));
  }
  if (dW) {
    ctx.template Alloc<T>(dW);
  }
  if (dX) {
    ctx.template Alloc<T>(dX);
  }

  // const T* x = X->data<T>();
  const T* dy = dO->data<T>();
  const T* w = W->data<T>();

  const T* ddx = nullptr;
  const T* ddw = nullptr;
  T *dw, *dx, *ddy;
  dw = dx = ddy = nullptr;
  T* transformed_dx = nullptr;
  std::vector<int> dilations = dilations_t;

  bool exhaustive_search = FLAGS_cudnn_exhaustive_search || exhaustive_search_t;
  bool deterministic = FLAGS_cudnn_deterministic;
  auto exhaustive_deterministic = exhaustive_search && deterministic;
  PADDLE_ENFORCE_EQ(exhaustive_deterministic,
                    false,
                    phi::errors::InvalidArgument(
                        "Cann't set exhaustive_search True and "
                        "FLAGS_cudnn_deterministic True at same time."));

  std::vector<int> paddings = paddings_t;

  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

  // transform Tensors to channel first-----------
  DenseTensor transformed_X_channel(X->type());
  DenseTensor transformed_dO_channel(dO->type());
  DenseTensor transformed_ddX_channel(X->type());

  DenseTensor transformed_ddO_channel(dO->type());
  DenseTensor transformed_dX_channel(X->type());

  if (channel_last) {
    ResizeToChannelFirst<Context, T>(ctx, X, &transformed_X_channel);
    TransToChannelFirst<Context, T>(ctx, X, &transformed_X_channel);

    ResizeToChannelFirst<Context, T>(ctx, dO, &transformed_dO_channel);
    TransToChannelFirst<Context, T>(ctx, dO, &transformed_dO_channel);

    if (ddX) {
      ResizeToChannelFirst<Context, T>(ctx, ddX, &transformed_ddX_channel);
      TransToChannelFirst<Context, T>(ctx, ddX, &transformed_ddX_channel);
    }

    if (ddO) {
      ResizeToChannelFirst<Context, T>(ctx, ddO, &transformed_ddO_channel);
    }
    if (dX) {
      ResizeToChannelFirst<Context, T>(ctx, dX, &transformed_dX_channel);
      ctx.template Alloc<T>(&transformed_dX_channel);
    }

  } else {
    transformed_X_channel = *X;
    transformed_dO_channel = *dO;
    if (ddX) {
      transformed_ddX_channel = *ddX;
    }
    if (ddO) {
      transformed_ddO_channel.ShareDataWith(*ddO);
    }
    if (dX) {
      transformed_dX_channel.ShareDataWith(*dX);
    }
  }

  auto in_dims = transformed_X_channel.dims();
  auto filter_dims = W->dims();
  DDim in_data_dims = slice_ddim(in_dims, 2, in_dims.size());
  DDim filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings, &dilations, padding_algorithm, in_data_dims, strides, ksize);

  int data_dim = strides.size();  // 2d or 3d
  bool is_sys_pad = funcs::IsSymmetricPadding(paddings, data_dim);
  DenseTensor transformed_X(X->type());
  DenseTensor transformed_ddX(X->type());

  DenseTensor transformed_dX(X->type());

  std::vector<int> padding_common(data_dim, 0);
  std::vector<int> input_pad(X->dims().size() * 2, 0);

  if (!is_sys_pad) {
    // get pad
    std::vector<int> padding_diff(data_dim);
    std::vector<int> new_input_shape_vec(data_dim + 2);
    new_input_shape_vec[0] = transformed_X_channel.dims()[0];
    new_input_shape_vec[1] = transformed_X_channel.dims()[1];

    for (size_t i = 0; i < data_dim; ++i) {
      padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
      padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
      new_input_shape_vec[i + 2] =
          transformed_X_channel.dims()[i + 2] + padding_diff[i];
      input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
      input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
    }
    DDim new_input_shape(make_ddim(new_input_shape_vec));
    transformed_X.Resize(new_input_shape);
    transformed_ddX.Resize(new_input_shape);
    transformed_dX.Resize(new_input_shape);

    ctx.template Alloc<T>(&transformed_X);

    if (ddX) {
      ctx.template Alloc<T>(&transformed_ddX);
    }
    if (dX) {
      ctx.template Alloc<T>(&transformed_dX);
    }

    // pad for input
    const int rank = X->dims().size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(
            ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
        if (ddX) {
          funcs::PadFunction<Context, T, 4>(ctx,
                                            input_pad,
                                            transformed_ddX_channel,
                                            pad_value,
                                            &transformed_ddX);
        }
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(
            ctx, input_pad, transformed_X_channel, pad_value, &transformed_X);
        if (ddX) {
          funcs::PadFunction<Context, T, 5>(ctx,
                                            input_pad,
                                            transformed_ddX_channel,
                                            pad_value,
                                            &transformed_ddX);
        }
      } break;
      default:
        PADDLE_THROW(phi::errors::InvalidArgument(
            "ConvOp only support tensors with 4 or 5 dimensions."));
    }

  } else {
    transformed_X.ShareDataWith(transformed_X_channel);
    if (ddX) {
      transformed_ddX.ShareDataWith(transformed_ddX_channel);
    }
    if (dX) {
      transformed_dX.ShareDataWith(transformed_dX_channel);
    }

    if (paddings.size() == data_dim) {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings[i];
      }
    } else {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings[2 * i];
      }
    }
  }

  const T* x = transformed_X.data<T>();

  int iwo_group = groups;
  int c_group = 1;
#if defined(PADDLE_WITH_HIP) || CUDNN_VERSION_MIN(7, 0, 1)
  iwo_group = 1;
  c_group = groups;
  groups = 1;
#endif
  auto dtype = paddle::platform::CudnnDataType<T>::type;

  auto handle = ctx.cudnn_handle();
  auto layout = paddle::platform::GetCudnnTensorFormat(
      paddle::platform::DataLayout::kNCHW);

  paddle::operators::ConvArgs args1{&transformed_ddX,
                                    W,
                                    &transformed_ddO_channel,
                                    strides,
                                    padding_common,
                                    dilations,
                                    dtype,
                                    groups,
                                    paddle::platform::DataLayout::kNCHW};
  paddle::operators::ConvArgs args2{&transformed_X,
                                    ddW,
                                    &transformed_ddO_channel,
                                    strides,
                                    padding_common,
                                    dilations,
                                    dtype,
                                    groups,
                                    paddle::platform::DataLayout::kNCHW};
  paddle::operators::ConvArgs args3{&transformed_ddX,
                                    dW,
                                    &transformed_dO_channel,
                                    strides,
                                    padding_common,
                                    dilations,
                                    dtype,
                                    groups,
                                    paddle::platform::DataLayout::kNCHW};
  paddle::operators::ConvArgs args4{&transformed_dX,
                                    ddW,
                                    &transformed_dO_channel,
                                    strides,
                                    padding_common,
                                    dilations,
                                    dtype,
                                    groups,
                                    paddle::platform::DataLayout::kNCHW};

#ifdef PADDLE_WITH_HIP
  paddle::operators::SearchResult<miopenConvFwdAlgorithm_t> fwd_result1;
  paddle::operators::SearchResult<miopenConvFwdAlgorithm_t> fwd_result2;
  paddle::operators::SearchResult<miopenConvBwdDataAlgorithm_t> data_result;
  paddle::operators::SearchResult<miopenConvBwdWeightsAlgorithm_t>
      filter_result;
#else
  paddle::operators::SearchResult<hipdnnConvolutionFwdAlgo_t> fwd_result1;
  paddle::operators::SearchResult<hipdnnConvolutionFwdAlgo_t> fwd_result2;
  paddle::operators::SearchResult<hipdnnConvolutionBwdDataAlgo_t> data_result;
  paddle::operators::SearchResult<hipdnnConvolutionBwdFilterAlgo_t>
      filter_result;
#endif

  // ddo = conv(ddI, W) + conv(I, ddW)
  size_t workspace_size = 0;

  T* transformed_ddy_channel = nullptr;
  if (ddO) {
    ddy = ddO->data<T>();
    transformed_ddy_channel = transformed_ddO_channel.data<T>();
    if (ddX) {
      args1.handle = handle;
      args1.idesc.set(transformed_ddX, iwo_group);
      args1.wdesc.set(*W, layout, iwo_group);
      args1.odesc.set(transformed_ddO_channel, iwo_group);
      args1.cdesc.set(dtype,
                      padding_common,
                      strides,
                      dilations,
                      paddle::platform::AllowTF32Cudnn(),
                      c_group);

#ifdef PADDLE_WITH_HIP
      using search1 =
          paddle::operators::SearchAlgorithm<miopenConvFwdAlgorithm_t>;
      workspace_size = search1::GetWorkspaceSize(args1);
      fwd_result1.algo = search1::Find<T>(
          args1, exhaustive_search, false, workspace_size, ctx);
#else
      using search1 =
          paddle::operators::SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
      fwd_result1 = search1::Find<T>(ctx, args1, exhaustive_search, false);
      workspace_size = search1::GetWorkspaceSize(args1, fwd_result1.algo);
#endif
    }

    if (ddW) {
      ddw = ddW->data<T>();
      args2.handle = handle;
      args2.idesc.set(transformed_X, iwo_group);
      args2.wdesc.set(*ddW, layout, iwo_group);
      args2.odesc.set(transformed_ddO_channel, iwo_group);
      args2.cdesc.set(dtype,
                      padding_common,
                      strides,
                      dilations,
                      paddle::platform::AllowTF32Cudnn(),
                      c_group);

#ifdef PADDLE_WITH_HIP
      using search2 =
          paddle::operators::SearchAlgorithm<miopenConvFwdAlgorithm_t>;
      workspace_size =
          std::max(workspace_size, search2::GetWorkspaceSize(args2));
      fwd_result2.algo = search2::Find<T>(
          args2, exhaustive_search, false, workspace_size, ctx);
#else
      using search2 =
          paddle::operators::SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
      fwd_result2 = search2::Find<T>(ctx, args2, exhaustive_search, false);
      workspace_size = std::max(
          workspace_size, search2::GetWorkspaceSize(args2, fwd_result2.algo));
#endif
    }
  }

  if (dW && ddX) {
    dw = dW->data<T>();
    args3.handle = handle;
    args3.idesc.set(transformed_ddX, iwo_group);
    args3.wdesc.set(*dW, layout, iwo_group);
    args3.odesc.set(transformed_dO_channel, iwo_group);
    args3.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations,
                    paddle::platform::AllowTF32Cudnn(),
                    c_group);

#ifdef PADDLE_WITH_HIP
    using search3 =
        paddle::operators::SearchAlgorithm<miopenConvBwdWeightsAlgorithm_t>;
    workspace_size = std::max(workspace_size, search3::GetWorkspaceSize(args3));
    filter_result.algo = search3::Find<T>(
        args3, exhaustive_search, deterministic, workspace_size, ctx);
#else
    using search3 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
    filter_result =
        search3::Find<T>(ctx, args3, exhaustive_search, deterministic);
    workspace_size = std::max(
        workspace_size, search3::GetWorkspaceSize(args3, filter_result.algo));
#endif
  }

  if (ddW && dX) {
    transformed_dx = transformed_dX.data<T>();

    args4.handle = handle;
    args4.idesc.set(transformed_dX, iwo_group);
    args4.wdesc.set(*ddW, layout, iwo_group);
    args4.odesc.set(transformed_dO_channel, iwo_group);
    args4.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations,
                    paddle::platform::AllowTF32Cudnn(),
                    c_group);

#ifdef PADDLE_WITH_HIP
    using search4 =
        paddle::operators::SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
    workspace_size = std::max(workspace_size, search4::GetWorkspaceSize(args4));
    data_result.algo = search4::Find<T>(
        args4, exhaustive_search, deterministic, workspace_size, ctx);
#else
    using search4 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
    data_result =
        search4::Find<T>(ctx, args4, exhaustive_search, deterministic);
    workspace_size = std::max(
        workspace_size, search4::GetWorkspaceSize(args4, data_result.algo));
#endif
  }

  int i_n, i_c, i_d, i_h, i_w;
  GetNCDHW(
      transformed_X.dims(), DataLayout::kNCHW, &i_n, &i_c, &i_d, &i_h, &i_w);

  int o_n, o_c, o_d, o_h, o_w;
  GetNCDHW(transformed_dO_channel.dims(),
           DataLayout::kNCHW,
           &o_n,
           &o_c,
           &o_d,
           &o_h,
           &o_w);

  int group_offset_in = i_c / groups * i_h * i_w * i_d;
  int group_offset_out = o_c / groups * o_h * o_w * o_d;
  int group_offset_filter = W->numel() / groups;

  paddle::operators::ScalingParamType<T> alpha = 1.0f;
  paddle::operators::ScalingParamType<T> beta = 0.0f;

  // NOTE(zhiqiu): inplace addto is not supportted in double grad yet.
  // ScalingParamType<T> beta = ctx.Attr<bool>("use_addto") ? 1.0f :
  // 0.0f;
  // VLOG(4) << "Conv_grad_grad: use_addto = " << ctx.Attr<bool>("use_addto");
  auto wkspace_handle = ctx.cudnn_workspace_handle();

  if (ddO) {
    if (ddX) {
      ddx = transformed_ddX.data<T>();
#ifdef PADDLE_WITH_HIP
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::miopenConvolutionForward(
                    handle,
                    &alpha,
                    args1.idesc.desc(),
                    ddx,
                    args1.wdesc.desc(),
                    w,
                    args1.cdesc.desc(),
                    fwd_result1.algo,
                    &beta,
                    args1.odesc.desc(),
                    transformed_ddy_channel,
                    workspace_ptr,
                    workspace_size));
          },
          workspace_size);
#else
      for (int i = 0; i < groups; i++) {
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_GPU_SUCCESS(
                  paddle::platform::dynload::hipdnnConvolutionForward(
                      handle,
                      &alpha,
                      args1.idesc.desc(),
                      ddx + i * group_offset_in,
                      args1.wdesc.desc(),
                      w + i * group_offset_filter,
                      args1.cdesc.desc(),
                      fwd_result1.algo,
                      workspace_ptr,
                      workspace_size,
                      &beta,
                      args1.odesc.desc(),
                      transformed_ddy_channel + i * group_offset_out));
            },
            workspace_size);
      }
#endif
    }
    if (ddW) {
#ifdef PADDLE_WITH_HIP
      // MIOPEN ONLY support beta to be 0.0f
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::miopenConvolutionForward(
                    handle,
                    &alpha,
                    args2.idesc.desc(),
                    x,
                    args2.wdesc.desc(),
                    ddw,
                    args2.cdesc.desc(),
                    fwd_result2.algo,
                    &beta,
                    args2.odesc.desc(),
                    transformed_ddy_channel,
                    workspace_ptr,
                    workspace_size));
          },
          workspace_size);
#else
      for (int i = 0; i < groups; i++) {
        wkspace_handle.RunFunc(
            [&](void* workspace_ptr) {
              PADDLE_ENFORCE_GPU_SUCCESS(
                  paddle::platform::dynload::hipdnnConvolutionForward(
                      handle,
                      &alpha,
                      args2.idesc.desc(),
                      x + i * group_offset_in,
                      args2.wdesc.desc(),
                      ddw + i * group_offset_filter,
                      args2.cdesc.desc(),
                      fwd_result2.algo,
                      workspace_ptr,
                      workspace_size,
                      &alpha,
                      args2.odesc.desc(),
                      transformed_ddy_channel + i * group_offset_out));
            },
            workspace_size);
      }
#endif
    }
    if (channel_last) {
      TransToChannelLast<Context, T>(ctx, &transformed_ddO_channel, ddO);
    }
  }
  T* transformed_dy_channel = transformed_dO_channel.data<T>();
  if (dW && ddX) {
    ddx = transformed_ddX.data<T>();
#ifdef PADDLE_WITH_HIP
    wkspace_handle.RunFunc(
        [&](void* workspace_ptr) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              paddle::platform::dynload::miopenConvolutionBackwardWeights(
                  handle,
                  &alpha,
                  args3.odesc.desc(),
                  transformed_dy_channel,
                  args3.idesc.desc(),
                  ddx,
                  args3.cdesc.desc(),
                  filter_result.algo,
                  &beta,
                  args3.wdesc.desc(),
                  dw,
                  workspace_ptr,
                  workspace_size));
        },
        workspace_size);
#else
    for (int i = 0; i < groups; i++) {
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::hipdnnConvolutionBackwardFilter(
                    handle,
                    &alpha,
                    args3.idesc.desc(),
                    ddx + i * group_offset_in,
                    args3.odesc.desc(),
                    transformed_dy_channel + i * group_offset_out,
                    args3.cdesc.desc(),
                    filter_result.algo,
                    workspace_ptr,
                    workspace_size,
                    &beta,
                    args3.wdesc.desc(),
                    dw + i * group_offset_filter));
          },
          workspace_size);
    }
#endif
  }

  if (dX && ddW) {
    ddw = ddW->data<T>();
#ifdef PADDLE_WITH_HIP
    wkspace_handle.RunFunc(
        [&](void* workspace_ptr) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              paddle::platform::dynload::miopenConvolutionBackwardData(
                  handle,
                  &alpha,
                  args4.odesc.desc(),
                  transformed_dy_channel,
                  args4.wdesc.desc(),
                  ddw,
                  args4.cdesc.desc(),
                  data_result.algo,
                  &beta,
                  args4.idesc.desc(),
                  transformed_dx,
                  workspace_ptr,
                  workspace_size));
        },
        workspace_size);
#else
    for (int i = 0; i < groups; i++) {
      wkspace_handle.RunFunc(
          [&](void* workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::hipdnnConvolutionBackwardData(
                    handle,
                    &alpha,
                    args4.wdesc.desc(),
                    ddw + i * group_offset_filter,
                    args4.odesc.desc(),
                    transformed_dy_channel + i * group_offset_out,
                    args4.cdesc.desc(),
                    data_result.algo,
                    workspace_ptr,
                    workspace_size,
                    &beta,
                    args4.idesc.desc(),
                    transformed_dx + i * group_offset_in));
          },
          workspace_size);
    }
#endif

    if (!is_sys_pad) {
      // reverse padded input
      std::vector<int> starts(X->dims().size(), 0);
      std::vector<int> axes(X->dims().size(), 0);

      for (size_t i = 0; i < X->dims().size(); ++i) {
        starts[i] = input_pad[2 * i];
        axes[i] = i;
      }
      if (X->dims().size() == 4) {
        paddle::operators::RemovePaddingSlice<Context, T, 4>(
            ctx, &transformed_dX, &transformed_dX_channel, starts, axes);
      } else {
        paddle::operators::RemovePaddingSlice<Context, T, 5>(
            ctx, &transformed_dX, &transformed_dX_channel, starts, axes);
      }
    }
    if (channel_last) {
      TransToChannelLast<Context, T>(ctx, &transformed_dX_channel, dX);
    }
  }
}

template <typename T, typename Context>
void DepthwiseConvDoubleGradGPUDNNKernel(
    const Context& ctx,
    const DenseTensor& input,
    const DenseTensor& filter,
    const DenseTensor& out_grad,
    const paddle::optional<DenseTensor>& input_grad_grad,
    const paddle::optional<DenseTensor>& filter_grad_grad,
    const std::vector<int>& strides,
    const std::vector<int>& paddings_t,
    const std::string& padding_algorithm,
    int groups,
    const std::vector<int>& dilations_t,
    const std::string& data_format,
    bool use_addto,
    int workspace_size_MB,
    bool exhaustive_search_t,
    bool fuse_relu,
    DenseTensor* input_grad,
    DenseTensor* filter_grad,
    DenseTensor* out_grad_grad) {
  ConvCudnnGradGradKernel<T>(ctx,
                             input,
                             filter,
                             out_grad,
                             input_grad_grad,
                             filter_grad_grad,
                             strides,
                             paddings_t,
                             padding_algorithm,
                             groups,
                             dilations_t,
                             data_format,
                             use_addto,
                             workspace_size_MB,
                             exhaustive_search_t,
                             input_grad,
                             filter_grad,
                             out_grad_grad);
}

template <typename T, typename Context>
void Conv3DCudnnGradGradKernel(
    const Context& ctx,
    const DenseTensor& input,
    const DenseTensor& filter,
    const DenseTensor& out_grad,
    const paddle::optional<DenseTensor>& input_grad_grad,
    const paddle::optional<DenseTensor>& filter_grad_grad,
    const std::vector<int>& strides,
    const std::vector<int>& paddings_t,
    const std::string& padding_algorithm,
    int groups,
    const std::vector<int>& dilations_t,
    const std::string& data_format,
    bool use_addto,
    int workspace_size_MB,
    bool exhaustive_search_t,
    DenseTensor* input_grad,
    DenseTensor* filter_grad,
    DenseTensor* out_grad_grad) {
  ConvCudnnGradGradKernel<T>(ctx,
                             input,
                             filter,
                             out_grad,
                             input_grad_grad,
                             filter_grad_grad,
                             strides,
                             paddings_t,
                             padding_algorithm,
                             groups,
                             dilations_t,
                             data_format,
                             use_addto,
                             workspace_size_MB,
                             exhaustive_search_t,
                             input_grad,
                             filter_grad,
                             out_grad_grad);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(conv2d_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnGradGradKernel,
                   float,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(conv3d_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnGradGradKernel,
                   float,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(depthwise_conv2d_grad_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::DepthwiseConvDoubleGradGPUDNNKernel,
                   float,
                   phi::dtype::float16) {}
#else
#if CUDNN_VERSION_MIN(8, 1, 0)
PD_REGISTER_KERNEL(conv2d_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnGradGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(conv3d_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnGradGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(depthwise_conv2d_grad_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::DepthwiseConvDoubleGradGPUDNNKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

#else

PD_REGISTER_KERNEL(conv2d_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnGradGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(conv3d_grad_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnGradGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(depthwise_conv2d_grad_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::DepthwiseConvDoubleGradGPUDNNKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

#endif

#endif
