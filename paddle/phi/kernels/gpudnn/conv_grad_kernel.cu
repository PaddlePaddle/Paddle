#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/conv_grad_kernel.h"

#include "paddle/fluid/framework/eigen.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/conv_miopen_helper.h"
#else
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#endif

#include "paddle/fluid/platform/cudnn_workspace_helper.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/fluid/platform/profiler.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/cpu/conv_util.h"
#include "paddle/phi/kernels/funcs/batch_norm_utils.h"
#include "paddle/phi/kernels/funcs/padding.h"
#include "paddle/phi/kernels/impl/conv_cudnn_impl.h"

namespace phi {

template <typename T, typename Context>
void ConvCudnnGradKernel(const Context& ctx,
                         const DenseTensor& input,
                         const DenseTensor& filter,
                         const DenseTensor& output_grad,
                         const std::vector<int>& strides_t,
                         const std::vector<int>& paddings_t,
                         const std::string& padding_algorithm,
                         int groups,
                         const std::vector<int>& dilations_t,
                         const std::string& data_format,
                         bool use_addto,
                         int workspace_size_MB,
                         bool exhaustive_search_t,
                         DenseTensor* input_grad,
                         DenseTensor* filter_grad) {
  if (input_grad) {
    ctx.template Alloc<T>(input_grad);
  }
  if (filter_grad) {
    ctx.template Alloc<T>(filter_grad);
  }

  std::vector<int> dilations = dilations_t;
  std::vector<int> strides = strides_t;
  std::vector<int> paddings = paddings_t;

  bool exhaustive_search = FLAGS_cudnn_exhaustive_search || exhaustive_search_t;
  bool deterministic = FLAGS_cudnn_deterministic;
  auto exhaustive_deterministic = exhaustive_search && deterministic;
  PADDLE_ENFORCE_EQ(exhaustive_deterministic,
                    false,
                    phi::errors::InvalidArgument(
                        "Cann't set exhaustive_search True and "
                        "FLAGS_cudnn_deterministic True at same time."));

  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

  auto dtype = paddle::platform::CudnnDataType<T>::type;

#ifdef PADDLE_WITH_HIP
  // HIP MIOPEN ONLY SUPPORT NCHW format
  auto compute_format = paddle::platform::DataLayout::kNCHW;
#else
  const bool compute_in_nhwc = dtype == HIPDNN_DATA_HALF && IsVoltaOrLater(ctx);
  auto compute_format = compute_in_nhwc && channel_last
                            ? paddle::platform::DataLayout::kNHWC
                            : paddle::platform::DataLayout::kNCHW;
#endif
  VLOG(3) << "Compute ConvGradOp with cuDNN:"
          << " data_format=" << data_format << " compute_format="
          << (compute_format == paddle::platform::DataLayout::kNHWC ? "NHWC"
                                                                    : "NCHW");

  // transform Tensor
  DenseTensor transformed_input_channel(input.type());
  DenseTensor transformed_output_grad_channel(output_grad.type());
  DenseTensor transformed_input_grad_channel(input.type());
  DenseTensor transformed_filter_channel(filter.type());
  DenseTensor transformed_filter_grad_channel(filter.type());

  if (channel_last && compute_format == paddle::platform::DataLayout::kNCHW) {
    VLOG(3) << "Transform input, output_grad, input_grad and tensor from "
               "NHWC to NCHW.";
    ResizeToChannelFirst<Context, T>(ctx, &input, &transformed_input_channel);
    TransToChannelFirst<Context, T>(ctx, &input, &transformed_input_channel);

    ResizeToChannelFirst<Context, T>(
        ctx, &output_grad, &transformed_output_grad_channel);
    TransToChannelFirst<Context, T>(
        ctx, &output_grad, &transformed_output_grad_channel);

    if (input_grad) {
      ResizeToChannelFirst<Context, T>(
          ctx, input_grad, &transformed_input_grad_channel);
      // NOTE(zhiqiu): If inplace_addto strategy is enabled, we need to copy
      // the data of input_grad to transformed_input_grad_channel.
      if (use_addto) {
        TransToChannelFirst<Context, T>(
            ctx, input_grad, &transformed_input_grad_channel);
      }
    }
  } else {
    transformed_input_channel.ShareDataWith(input);
    transformed_output_grad_channel.ShareDataWith(output_grad);
    if (input_grad) {
      transformed_input_grad_channel.ShareDataWith(*input_grad);
    }
  }

  if (compute_format == paddle::platform::DataLayout::kNHWC) {
    VLOG(3) << "Transform filter and filter_grad tensor from NCHW to NHWC.";
    ResizeToChannelLast<Context, T>(ctx, &filter, &transformed_filter_channel);
    TransToChannelLast<Context, T>(ctx, &filter, &transformed_filter_channel);

    if (filter_grad) {
      ResizeToChannelLast<Context, T>(
          ctx, filter_grad, &transformed_filter_grad_channel);
    }
  } else {
    transformed_filter_channel.ShareDataWith(filter);
    if (filter_grad) {
      transformed_filter_grad_channel.ShareDataWith(*filter_grad);
    }
  }

  //  update paddings
  auto in_dims = transformed_input_channel.dims();
  auto filter_dims = transformed_filter_channel.dims();
  DDim in_data_dims;
  DDim filter_data_dims;
  if (compute_format == paddle::platform::DataLayout::kNCHW) {
    in_data_dims = slice_ddim(in_dims, 2, in_dims.size());
    filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  } else {
    in_data_dims = slice_ddim(in_dims, 1, in_dims.size() - 1);
    filter_data_dims = slice_ddim(filter_dims, 1, filter_dims.size() - 1);
  }
  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings, &dilations, padding_algorithm, in_data_dims, strides, ksize);

  // cuDNN only supports padding the same amount on every dimension.
  // So we create a new padded input tensor.
  int data_dim = strides.size();  // 2d or 3d
  bool is_sys_pad = funcs::IsSymmetricPadding(paddings, data_dim);
  Tensor transformed_input(input.type());
  Tensor transformed_input_grad(input.type());
  std::vector<int> padding_common(data_dim, 0);
  std::vector<int> input_pad(transformed_input_channel.dims().size() * 2, 0);

  if (!is_sys_pad) {
    // get pad
    std::vector<int> padding_diff(data_dim);
    std::vector<int> new_input_shape_vec(data_dim + 2);
    new_input_shape_vec[0] = transformed_input_channel.dims()[0];
    if (compute_format == paddle::platform::DataLayout::kNCHW) {
      new_input_shape_vec[1] = transformed_input_channel.dims()[1];
    } else {
      new_input_shape_vec[data_dim + 1] =
          transformed_input_channel.dims()[data_dim + 1];
    }

    for (size_t i = 0; i < data_dim; ++i) {
      padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
      padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
      if (compute_format == paddle::platform::DataLayout::kNCHW) {
        new_input_shape_vec[i + 2] =
            transformed_input_channel.dims()[i + 2] + padding_diff[i];
      } else {
        new_input_shape_vec[i + 1] =
            transformed_input_channel.dims()[i + 1] + padding_diff[i];
      }
      if (compute_format == paddle::platform::DataLayout::kNCHW) {
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      } else {
        input_pad[2 * i + 2] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 2 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
    }
    DDim new_input_shape(make_ddim(new_input_shape_vec));
    transformed_input.Resize(new_input_shape);
    ctx.template Alloc<T>(&transformed_input);

    transformed_input_grad.Resize(new_input_shape);

    if (input_grad) {
      ctx.template Alloc<T>(&transformed_input_grad);
    }
    // pad for input
    const int rank = transformed_input_channel.dims().size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(ctx,
                                          input_pad,
                                          transformed_input_channel,
                                          pad_value,
                                          &transformed_input);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(ctx,
                                          input_pad,
                                          transformed_input_channel,
                                          pad_value,
                                          &transformed_input);
      } break;
      default:
        PADDLE_THROW(phi::errors::InvalidArgument(
            "ConvOp only support tensors with 4 or 5 dimensions."));
    }
  } else {
    transformed_input.ShareDataWith(transformed_input_channel);
    if (input_grad) {
      transformed_input_grad.ShareDataWith(transformed_input_grad_channel);
    }
    if (paddings.size() == data_dim) {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings[i];
      }
    } else {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings[2 * i];
      }
    }
  }

  const T* input_data = transformed_input.data<T>();
  const T* output_grad_data = transformed_output_grad_channel.data<T>();
  const T* filter_data = transformed_filter_channel.data<T>();
  T* filter_grad_data = nullptr;
  T* input_grad_data = nullptr;
  T* transformed_input_grad_data = nullptr;

  paddle::platform::DataLayout layout =
      compute_format == paddle::platform::DataLayout::kNHWC
          ? paddle::platform::DataLayout::kNHWC
          : paddle::platform::DataLayout::kNCHW;

  paddle::operators::ConvArgs args1{&transformed_input_grad,
                                    &transformed_filter_channel,
                                    &transformed_output_grad_channel,
                                    strides,
                                    padding_common,
                                    dilations,
                                    dtype,
                                    groups,
                                    layout};
  paddle::operators::ConvArgs args2{&transformed_input,
                                    &transformed_filter_grad_channel,
                                    &transformed_output_grad_channel,
                                    strides,
                                    padding_common,
                                    dilations,
                                    dtype,
                                    groups,
                                    layout};

  auto handle = ctx.cudnn_handle();
  // TODO(phlrain): replace paddle::platform::DataLaytout to phi::DataLayout

  if (transformed_input.dims().size() == 5) {
    layout = compute_format == paddle::platform::DataLayout::kNHWC
                 ? paddle::platform::DataLayout::kNDHWC
                 : paddle::platform::DataLayout::kNCDHW;
  }
  auto layout_tensor = paddle::platform::GetCudnnTensorFormat(layout);
  auto workspace_handle = ctx.cudnn_workspace_handle();

  int i_n, i_c, i_d, i_h, i_w;
  int o_n, o_c, o_d, o_h, o_w;
  if (compute_format == paddle::platform::DataLayout::kNHWC) {
    paddle::operators::GetNCDHW(transformed_input.dims(),
                                paddle::platform::DataLayout::kNHWC,
                                &i_n,
                                &i_c,
                                &i_d,
                                &i_h,
                                &i_w);
    paddle::operators::GetNCDHW(transformed_output_grad_channel.dims(),
                                paddle::platform::DataLayout::kNHWC,
                                &o_n,
                                &o_c,
                                &o_d,
                                &o_h,
                                &o_w);
  } else {
    paddle::operators::GetNCDHW(transformed_input.dims(),
                                paddle::platform::DataLayout::kNCHW,
                                &i_n,
                                &i_c,
                                &i_d,
                                &i_h,
                                &i_w);
    paddle::operators::GetNCDHW(transformed_output_grad_channel.dims(),
                                paddle::platform::DataLayout::kNCHW,
                                &o_n,
                                &o_c,
                                &o_d,
                                &o_h,
                                &o_w);
  }

  int group_offset_in = i_c / groups * i_h * i_w * i_d;
  int group_offset_out = o_c / groups * o_h * o_w * o_d;
  int group_offset_filter = transformed_filter_channel.numel() / groups;

// ------------------- cudnn backward algorithm ---------------------
#ifdef PADDLE_WITH_HIP
  paddle::operators::SearchResult<miopenConvBwdDataAlgorithm_t> bwd_result;
  paddle::operators::SearchResult<miopenConvBwdWeightsAlgorithm_t>
      filter_result;
#else
  paddle::operators::SearchResult<hipdnnConvolutionBwdDataAlgo_t> bwd_result;
  paddle::operators::SearchResult<hipdnnConvolutionBwdFilterAlgo_t>
      filter_result;
#endif
  // input data workspace_size
  size_t workspace_size_d = 0;
  // weight workspace_size
  size_t workspace_size_w = 0;
  int iwo_groups = groups;
  int c_groups = 1;

#if defined(PADDLE_WITH_HIP) || CUDNN_VERSION_MIN(7, 0, 1)
  iwo_groups = 1;
  c_groups = groups;
  groups = 1;
#endif

  if (input_grad) {
    // ------------------- cudnn descriptors ---------------------
    input_grad_data = input_grad->data<T>();
    transformed_input_grad_data = transformed_input_grad.data<T>();

    args1.handle = handle;
    args1.idesc.set(transformed_input_grad, layout_tensor);
    args1.wdesc.set(transformed_filter_channel, layout_tensor, iwo_groups);
    args1.odesc.set(transformed_output_grad_channel, layout_tensor);
    args1.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations,
                    paddle::platform::AllowTF32Cudnn(),
                    c_groups);

#ifdef PADDLE_WITH_HIP
    using search1 =
        paddle::operators::SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
    workspace_size_d =
        std::max(workspace_size_d, search1::GetWorkspaceSize(args1));
    bwd_result.algo = search1::Find<T>(
        args1, exhaustive_search, deterministic, workspace_size_d, ctx);
#else
    using search1 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
    bwd_result = search1::Find<T>(ctx, args1, exhaustive_search, deterministic);
    workspace_size_d = std::max(workspace_size_d, bwd_result.workspace_size);
#endif
  }

  if (filter_grad) {
    // ------------------- cudnn descriptors ---------------------
    filter_grad_data = transformed_filter_grad_channel.data<T>();
    args2.handle = handle;
    args2.idesc.set(transformed_input, layout_tensor);
    args2.wdesc.set(transformed_filter_grad_channel, layout_tensor, iwo_groups);
    args2.odesc.set(transformed_output_grad_channel, layout_tensor);
    args2.cdesc.set(dtype,
                    padding_common,
                    strides,
                    dilations,
                    paddle::platform::AllowTF32Cudnn(),
                    c_groups);
#ifdef PADDLE_WITH_HIP
    using search2 =
        paddle::operators::SearchAlgorithm<miopenConvBwdWeightsAlgorithm_t>;
    workspace_size_w =
        std::max(workspace_size_w, search2::GetWorkspaceSize(args2));
    filter_result.algo = search2::Find<T>(
        args2, exhaustive_search, deterministic, workspace_size_w, ctx);
#else
    using search2 =
        paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdFilterAlgoPerf_t>;
    filter_result =
        search2::Find<T>(ctx, args2, exhaustive_search, deterministic);
    VLOG(3) << "filter algo: " << filter_result.algo << ", time "
            << filter_result.time;
    workspace_size_w = std::max(workspace_size_w, filter_result.workspace_size);
#endif
  }

  // ------------------- cudnn conv backward data ---------------------
  paddle::operators::ScalingParamType<T> alpha = 1.0f;
#ifdef PADDLE_WITH_HIP
  // MIOPEN ONLY support beta to be 0.0f
  paddle::operators::ScalingParamType<T> beta = 0.0f;
#else
  paddle::operators::ScalingParamType<T> beta = use_addto ? 1.0f : 0.0f;

#endif
  VLOG(4) << "Conv_grad: use_addto = " << use_addto;

  if (input_grad) {
// When beta is 0, it is unnecessary to reset input_grad.
// When beta is 1, the output cannot be reset since addt strategy used.
#ifdef PADDLE_WITH_HIP
    if (use_addto) {
      DenseTensor temp_tensor(transformed_input_grad.type());
      temp_tensor.Resize(transformed_input_grad.dims());
      T* temp_tensor_data = ctx.template Alloc<T>(&temp_tensor);
      workspace_handle.RunFunc(
          [&](void* cudnn_workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::miopenConvolutionBackwardData(
                    handle,
                    &alpha,
                    args1.odesc.desc(),
                    output_grad_data,
                    args1.wdesc.desc(),
                    filter_data,
                    args1.cdesc.desc(),
                    bwd_result.algo,
                    &beta,
                    args1.idesc.desc(),
                    temp_tensor_data,
                    cudnn_workspace_ptr,
                    workspace_size_d));
          },
          workspace_size_d);
      PADDLE_ENFORCE_GPU_SUCCESS(paddle::platform::dynload::miopenOpTensor(
          handle,
          miopenTensorOpAdd,
          &alpha,
          args1.idesc.desc(),
          transformed_input_grad_data,
          &alpha,
          args1.idesc.desc(),
          temp_tensor_data,
          &beta,
          args1.idesc.desc(),
          transformed_input_grad_data));
    } else {
      workspace_handle.RunFunc(
          [&](void* cudnn_workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::miopenConvolutionBackwardData(
                    handle,
                    &alpha,
                    args1.odesc.desc(),
                    output_grad_data,
                    args1.wdesc.desc(),
                    filter_data,
                    args1.cdesc.desc(),
                    bwd_result.algo,
                    &beta,
                    args1.idesc.desc(),
                    transformed_input_grad_data,
                    cudnn_workspace_ptr,
                    workspace_size_d));
          },
          workspace_size_d);
    }

#else
    for (int i = 0; i < groups; i++) {
      workspace_handle.RunFunc(
          [&](void* cudnn_workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::hipdnnConvolutionBackwardData(
                    handle,
                    &alpha,
                    args1.wdesc.desc(),
                    filter_data + i * group_offset_filter,
                    args1.odesc.desc(),
                    output_grad_data + i * group_offset_out,
                    args1.cdesc.desc(),
                    bwd_result.algo,
                    cudnn_workspace_ptr,
                    workspace_size_d,
                    &beta,
                    args1.idesc.desc(),
                    transformed_input_grad_data + i * group_offset_in));
          },
          workspace_size_d);
    }
#endif
    if (!is_sys_pad) {
      std::vector<int> starts(transformed_input_channel.dims().size(), 0);
      std::vector<int> axes(transformed_input_channel.dims().size(), 0);

      for (size_t i = 0; i < transformed_input_channel.dims().size(); ++i) {
        starts[i] = input_pad[2 * i];
        axes[i] = i;
      }

      ctx.template Alloc<T>(&transformed_input_grad_channel);
      if (transformed_input_channel.dims().size() == 4) {
        paddle::operators::RemovePaddingSlice<Context, T, 4>(
            ctx,
            &transformed_input_grad,
            &transformed_input_grad_channel,
            starts,
            axes);
      } else {
        paddle::operators::RemovePaddingSlice<Context, T, 5>(
            ctx,
            &transformed_input_grad,
            &transformed_input_grad_channel,
            starts,
            axes);
      }
    }

    if (channel_last && compute_format == paddle::platform::DataLayout::kNCHW) {
      TransToChannelLast<Context, T>(
          ctx, &transformed_input_grad_channel, input_grad);
    }
  }

  // filter_grad do not use inplace addto.
  paddle::operators::ScalingParamType<T> beta_filter = 0.0f;
  // ------------------- cudnn conv backward filter ---------------------
  if (filter_grad) {
// Because beta is zero, it is unnecessary to reset filter_grad.
#ifdef PADDLE_WITH_HIP
    workspace_handle.RunFunc(
        [&](void* cudnn_workspace_ptr) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              paddle::platform::dynload::miopenConvolutionBackwardWeights(
                  handle,
                  &alpha,
                  args2.odesc.desc(),
                  output_grad_data,
                  args2.idesc.desc(),
                  input_data,
                  args2.cdesc.desc(),
                  filter_result.algo,
                  &beta,
                  args2.wdesc.desc(),
                  filter_grad_data,
                  cudnn_workspace_ptr,
                  workspace_size_w));
        },
        workspace_size_w);
#else
    for (int i = 0; i < groups; i++) {
      workspace_handle.RunFunc(
          [&](void* cudnn_workspace_ptr) {
            PADDLE_ENFORCE_GPU_SUCCESS(
                paddle::platform::dynload::hipdnnConvolutionBackwardFilter(
                    handle,
                    &alpha,
                    args2.idesc.desc(),
                    input_data + i * group_offset_in,
                    args2.odesc.desc(),
                    output_grad_data + i * group_offset_out,
                    args2.cdesc.desc(),
                    filter_result.algo,
                    cudnn_workspace_ptr,
                    workspace_size_w,
                    &beta_filter,
                    args2.wdesc.desc(),
                    filter_grad_data + i * group_offset_filter));
          },
          workspace_size_w);
    }
#endif

    if (compute_format == paddle::platform::DataLayout::kNHWC) {
      TransToChannelFirst<Context, T>(
          ctx, &transformed_filter_grad_channel, filter_grad);
    }
  }
}

template <typename T, typename Context>
void Conv3DCudnnGradKernel(const Context& dev_ctx,
                           const DenseTensor& input,
                           const DenseTensor& filter,
                           const DenseTensor& out_grad,
                           const std::vector<int>& strides,
                           const std::vector<int>& paddings,
                           const std::string& paddding_algorithm,
                           int groups,
                           const std::vector<int>& dilations,
                           const std::string& data_format,
                           bool use_addto,
                           int workspace_size_MB,
                           bool exhaustive_search,
                           DenseTensor* input_grad,
                           DenseTensor* filter_grad) {
  ConvCudnnGradKernel<T>(dev_ctx,
                         input,
                         filter,
                         out_grad,
                         strides,
                         paddings,
                         paddding_algorithm,
                         groups,
                         dilations,
                         data_format,
                         use_addto,
                         workspace_size_MB,
                         exhaustive_search,
                         input_grad,
                         filter_grad);
}

template <typename T, typename Context>
void DepthwiseConvCudnnGradKernel(const Context& dev_ctx,
                                  const DenseTensor& input,
                                  const DenseTensor& filter,
                                  const DenseTensor& out_grad,
                                  const std::vector<int>& strides,
                                  const std::vector<int>& paddings,
                                  const std::string& paddding_algorithm,
                                  int groups,
                                  const std::vector<int>& dilations,
                                  const std::string& data_format,
                                  bool use_addto,
                                  int workspace_size_MB,
                                  bool exhaustive_search,
                                  bool fuse_relu,
                                  DenseTensor* input_grad,
                                  DenseTensor* filter_grad) {
  ConvCudnnGradKernel<T>(dev_ctx,
                         input,
                         filter,
                         out_grad,
                         strides,
                         paddings,
                         paddding_algorithm,
                         groups,
                         dilations,
                         data_format,
                         use_addto,
                         workspace_size_MB,
                         exhaustive_search,
                         input_grad,
                         filter_grad);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(conv2d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnGradKernel,
                   float,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(conv3d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnGradKernel,
                   float,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(depthwise_conv2d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::DepthwiseConvCudnnGradKernel,
                   float,
                   phi::dtype::float16) {}
#else
#if CUDNN_VERSION_MIN(8, 1, 0)
PD_REGISTER_KERNEL(conv2d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(conv3d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnGradKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#else
PD_REGISTER_KERNEL(conv2d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(conv3d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

#endif

#endif
