// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifndef PADDLE_WITH_HIP

#include "paddle/phi/kernels/affine_grid_grad_kernel.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/platform/device/gpu/gpu_device_function.h"
#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/fluid/platform/device/gpu/gpu_info.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/int_array.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

using ScopedSpatialTransformerDescriptor =
    paddle::platform::ScopedSpatialTransformerDescriptor;

template <typename T, typename Context>
void AffineGridGradCudnnKernel(const Context& dev_ctx,
                               const DenseTensor& output_grad,
                               const IntArray& outputShape,
                               bool align_corners,
                               DenseTensor* input_grad) {
  PADDLE_ENFORCE_EQ(
      paddle::platform::is_gpu_place(dev_ctx.GetPlace()),
      true,
      phi::errors::InvalidArgument(
          "Only support for CUDAPlace.Please switch your context from "
          "CPUPlace to CUDAPlace or update your cudnn."));
  auto handle = dev_ctx.cudnn_handle();
  auto& theta_grad = input_grad;

  int n = output_grad.dims()[0];
  auto& size_attr = outputShape.GetData();
  int h_size_data[4] = {0};
  h_size_data[0] = n;
  h_size_data[1] = size_attr[1];
  h_size_data[2] = size_attr[2];
  h_size_data[3] = size_attr[3];

  ScopedSpatialTransformerDescriptor st_desc;
  cudnnSpatialTransformerDescriptor_t cudnn_st_desc =
      st_desc.descriptor<T>(4, h_size_data);

  const T* output_grad_data = output_grad.data<T>();
  T* theta_grad_data = dev_ctx.template Alloc<T>(theta_grad);

  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::cudnnSpatialTfGridGeneratorBackward(
          handle, cudnn_st_desc, output_grad_data, theta_grad_data));
}

}  // namespace phi

PD_REGISTER_KERNEL(affine_grid_grad,  // cuda_only
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::AffineGridGradCudnnKernel,
                   float,
                   double){};
#endif
