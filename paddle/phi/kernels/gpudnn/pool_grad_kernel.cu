#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/pool_grad_kernel.h"

#include "paddle/fluid/platform/device/gpu/gpu_dnn.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/funcs/pooling.h"
#include "paddle/phi/kernels/gpudnn/pool_gpudnn.h"
#include "paddle/phi/kernels/pool_kernel.h"

#ifdef PADDLE_WITH_HIP
#include "paddle/phi/kernels/impl/pool_grad_kernel_impl.h"  //  PoolGradRawGPUDNNKernel will call PoolGradRawKernel for pooling type "max" in ROCm
#endif

namespace phi {

template <typename T, typename Context>
void PoolGradRawGPUDNNKernel(const Context& ctx,
                             const DenseTensor& x,
                             const DenseTensor& out,
                             const DenseTensor& dout,
                             const std::vector<int>& kernel_size,
                             const std::vector<int>& strides,
                             const std::vector<int>& paddings,
                             bool exclusive,
                             const std::string& data_format,
                             const std::string& pooling_type,
                             bool global_pooling,
                             bool adaptive,
                             const std::string& padding_algorithm,
                             DenseTensor* dx) {
  PADDLE_ENFORCE_EQ(
      paddle::platform::is_gpu_place(ctx.GetPlace()),
      true,
      errors::InvalidArgument("Pool operator CUDA kernel must use CUDAPlace "
                              "rather than CPUPlace."));

  const DenseTensor* input = &x;
  const DenseTensor* output = &out;
  const DenseTensor* output_grad = &dout;
  DenseTensor* input_grad = dx;
  std::vector<int> paddings_ = paddings;
  std::vector<int> kernel_size_ = kernel_size;

  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");

#ifdef PADDLE_WITH_HIP
  if (pooling_type == "max") {
    PoolGradRawKernel<T, GPUContext>(ctx,
                                     x,
                                     out,
                                     dout,
                                     kernel_size,
                                     strides,
                                     paddings_,
                                     exclusive,
                                     data_format,
                                     pooling_type,
                                     global_pooling,
                                     adaptive,
                                     padding_algorithm,
                                     dx);
    return;
  }
#endif

  // update paddings
  auto in_x_dims = input->dims();
  DDim data_dims;
  if (channel_last) {
    data_dims = slice_ddim(in_x_dims, 1, in_x_dims.size() - 1);
  } else {
    data_dims = slice_ddim(in_x_dims, 2, in_x_dims.size());
  }
  funcs::UpdatePadding(&paddings_,
                       global_pooling,
                       adaptive,
                       padding_algorithm,
                       data_dims,
                       strides,
                       kernel_size_);
  if (data_dims.size() * 2 == static_cast<int>(paddings_.size())) {
    for (int i = 0; i < data_dims.size(); ++i) {
      paddings_.erase(paddings_.begin() + i + 1);
    }
  }

  if (global_pooling) {
    funcs::UpdateKernelSize(&kernel_size_, data_dims);
  }

  // ------- tensor grad --------------
  DenseTensor transformed_input(input->type());
  DenseTensor transformed_output(output->type());
  DenseTensor transformed_output_grad(output_grad->type());

  ctx.template Alloc<T>(input_grad);
  DenseTensor transformed_input_grad(input_grad->type());
  GPUDNNDataLayout layout;
  const std::string str_NCHW = "NCHW", str_NHWC = "NHWC";
  const std::string str_NCDHW = "NCDHW", str_NDHWC = "NDHWC";
  if (data_format == str_NDHWC) {
    layout = GPUDNNDataLayout::kNCDHW;
    std::vector<int> axis{0, 4, 1, 2, 3};

    // input
    transformed_input.Resize(input->dims());
    auto in_dims_vec = vectorize(input->dims());
    in_dims_vec[1] = input->dims()[4];
    in_dims_vec[2] = input->dims()[1];
    in_dims_vec[3] = input->dims()[2];
    in_dims_vec[4] = input->dims()[3];
    transformed_input.Resize(make_ddim(in_dims_vec));
    ctx.Alloc(&transformed_input, input->type());

    funcs::Transpose<Context, T, 5> trans5;
    trans5(ctx, *input, &transformed_input, axis);

    // output
    transformed_output.Resize(output->dims());
    auto out_dims_vec = vectorize(output->dims());
    out_dims_vec[1] = output->dims()[4];
    out_dims_vec[2] = output->dims()[1];
    out_dims_vec[3] = output->dims()[2];
    out_dims_vec[4] = output->dims()[3];
    transformed_output.Resize(make_ddim(out_dims_vec));

    ctx.Alloc(&transformed_output, output->type());

    funcs::Transpose<Context, T, 5> trans5_v2;
    trans5_v2(ctx, *output, &transformed_output, axis);

    // output grad
    transformed_output_grad.Resize(make_ddim(out_dims_vec));
    ctx.Alloc(&transformed_output_grad, output_grad->type());

    funcs::Transpose<Context, T, 5> trans5_v3;
    trans5_v3(ctx, *output_grad, &transformed_output_grad, axis);

    // input grad
    transformed_input_grad.Resize(make_ddim(in_dims_vec));

#ifdef PADDLE_WITH_HIP
    // MIOPEN not support NHWC data layout
  } else if (data_format == str_NHWC) {
    layout = GPUDNNDataLayout::kNCHW;

    std::vector<int> axis{0, 3, 1, 2};

    // input
    transformed_input.Resize(input->dims());
    auto in_dims_vec = vectorize(input->dims());
    in_dims_vec[1] = input->dims()[3];
    in_dims_vec[2] = input->dims()[1];
    in_dims_vec[3] = input->dims()[2];
    transformed_input.Resize(make_ddim(in_dims_vec));
    ctx.Alloc(&transformed_input, input->type());

    funcs::Transpose<Context, T, 4> trans4;
    trans4(ctx, *input, &transformed_input, axis);

    // output
    transformed_output.Resize(output->dims());
    auto out_dims_vec = vectorize(output->dims());
    out_dims_vec[1] = output->dims()[3];
    out_dims_vec[2] = output->dims()[1];
    out_dims_vec[3] = output->dims()[2];
    transformed_output.Resize(make_ddim(out_dims_vec));
    ctx.Alloc(&transformed_output, output->type());

    funcs::Transpose<Context, T, 4> trans4_v2;
    trans4_v2(ctx, *output, &transformed_output, axis);

    // output grad
    transformed_output_grad.Resize(make_ddim(out_dims_vec));
    ctx.Alloc(&transformed_output_grad, output_grad->type());

    funcs::Transpose<Context, T, 4> trans4_v3;
    trans4_v3(ctx, *output_grad, &transformed_output_grad, axis);

    // input grad
    transformed_input_grad.Resize(make_ddim(in_dims_vec));
#endif
  } else {
    layout = GetLayoutFromStr(data_format);
    transformed_input = *input;
    transformed_output = *output;
    transformed_output_grad = *output_grad;
    transformed_input_grad = *input_grad;
  }

  const T* input_data = transformed_input.data<T>();
  const T* output_data = transformed_output.data<T>();
  const T* output_grad_data = transformed_output_grad.data<T>();

  // ------------------- cudnn descriptors ---------------------
  ScopedTensorDescriptor input_desc;
  ScopedTensorDescriptor output_desc;
  ScopedPoolingDescriptor pool_desc;

#ifdef PADDLE_WITH_HIP
  miopenTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
      layout, vectorize<int>(transformed_input.dims()));
  miopenTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
      layout, vectorize<int>(transformed_output.dims()));
#else
  hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
      layout, vectorize<int>(transformed_input.dims()));
  hipdnnTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
      layout, vectorize<int>(transformed_output.dims()));
#endif
  PoolingMode pooling_mode;
  if (pooling_type == "max") {
    if (FLAGS_cudnn_deterministic) {
      pooling_mode = PoolingMode::kMaximumDeterministic;
    } else {
      pooling_mode = PoolingMode::kMaximum;
    }
  } else {
    pooling_mode = exclusive ? PoolingMode::kAverageExclusive
                             : PoolingMode::kAverageInclusive;
  }

#ifdef PADDLE_WITH_HIP
  miopenPoolingDescriptor_t cudnn_pool_desc =
      pool_desc.descriptor(pooling_mode, kernel_size_, paddings_, strides);
#else
  hipdnnPoolingDescriptor_t cudnn_pool_desc =
      pool_desc.descriptor(pooling_mode, kernel_size_, paddings_, strides);
#endif

  // ------------------- cudnn pool algorithm ---------------------
  auto handle = ctx.cudnn_handle();
  ScalingParamType<T> alpha = 1.0f, beta = 0.0f;
  if (input_grad) {
    T* input_grad_data = ctx.template Alloc<T>(&transformed_input_grad);
// Because beta is zero, it is unnecessary to reset input_grad.
#ifdef PADDLE_WITH_HIP
    char* pool_workspace;
    size_t pool_worksize = 0;
    PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenPoolingGetWorkSpaceSizeV2(
        cudnn_pool_desc, cudnn_output_desc, &pool_worksize));
    PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(&pool_workspace, pool_worksize));
    PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenPoolingBackward(handle,
                                                              cudnn_pool_desc,
                                                              &alpha,
                                                              cudnn_output_desc,
                                                              output_data,
                                                              cudnn_output_desc,
                                                              output_grad_data,
                                                              cudnn_input_desc,
                                                              input_data,
                                                              &beta,
                                                              cudnn_input_desc,
                                                              input_grad_data,
                                                              pool_workspace));
    PADDLE_ENFORCE_GPU_SUCCESS(hipFree(pool_workspace));
#else
    PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnPoolingBackward(handle,
                                                             cudnn_pool_desc,
                                                             &alpha,
                                                             cudnn_output_desc,
                                                             output_data,
                                                             cudnn_output_desc,
                                                             output_grad_data,
                                                             cudnn_input_desc,
                                                             input_data,
                                                             &beta,
                                                             cudnn_input_desc,
                                                             input_grad_data));
#endif

    if (data_format == str_NDHWC) {
      std::vector<int> axis{0, 2, 3, 4, 1};
      funcs::Transpose<Context, T, 5> trans5_v4;
      trans5_v4(ctx, transformed_input_grad, input_grad, axis);
    }
#ifdef PADDLE_WITH_HIP
    // MIOPEN not support NHWC data layout
    if (data_format == str_NHWC) {
      std::vector<int> axis{0, 2, 3, 1};
      funcs::Transpose<Context, T, 4> trans4_v4;
      trans4_v4(ctx, transformed_input_grad, input_grad, axis);
    }
#endif
  }
}

template <typename T, typename Context>
void Pool2dGradGPUDNNKernel(const Context& ctx,
                            const DenseTensor& x,
                            const DenseTensor& out,
                            const DenseTensor& dout,
                            const IntArray& kernel_size,
                            const std::vector<int>& strides,
                            const std::vector<int>& paddings,
                            bool ceil_mode,
                            bool exclusive,
                            const std::string& data_format,
                            const std::string& pooling_type,
                            bool global_pooling,
                            bool adaptive,
                            const std::string& padding_algorithm,
                            DenseTensor* dx) {
  std::vector<int> kernel_size_val(kernel_size.GetData().begin(),
                                   kernel_size.GetData().end());
  PoolGradRawGPUDNNKernel<T, Context>(ctx,
                                      x,
                                      out,
                                      dout,
                                      kernel_size_val,
                                      strides,
                                      paddings,
                                      exclusive,
                                      data_format,
                                      pooling_type,
                                      global_pooling,
                                      adaptive,
                                      padding_algorithm,
                                      dx);
}

template <typename T, typename Context>
void Pool2dDoubleGradGPUDNNKernel(const Context& ctx,
                                  const DenseTensor& x,
                                  const IntArray& kernel_size,
                                  const std::vector<int>& strides,
                                  const std::vector<int>& paddings,
                                  bool ceil_mode,
                                  bool exclusive,
                                  const std::string& data_format,
                                  const std::string& pooling_type,
                                  bool global_pooling,
                                  bool adaptive,
                                  const std::string& padding_algorithm,
                                  DenseTensor* out) {
  if (pooling_type == "max") {
    PADDLE_THROW(
        errors::InvalidArgument("Pool op grad grad only supports avgpool."));
  } else {
    Pool2dGPUDNNKernel<T, Context>(ctx,
                                   x,
                                   kernel_size,
                                   strides,
                                   paddings,
                                   ceil_mode,
                                   exclusive,
                                   data_format,
                                   pooling_type,
                                   global_pooling,
                                   adaptive,
                                   padding_algorithm,
                                   out);
  }
}

template <typename T, typename Context>
void Pool3dGradGPUDNNKernel(const Context& ctx,
                            const DenseTensor& x,
                            const DenseTensor& out,
                            const DenseTensor& dout,
                            const std::vector<int>& kernel_size,
                            const std::vector<int>& strides,
                            const std::vector<int>& paddings,
                            bool ceil_mode,
                            bool exclusive,
                            const std::string& data_format,
                            const std::string& pooling_type,
                            bool global_pooling,
                            bool adaptive,
                            const std::string& padding_algorithm,
                            DenseTensor* dx) {
  PoolGradRawGPUDNNKernel<T, Context>(ctx,
                                      x,
                                      out,
                                      dout,
                                      kernel_size,
                                      strides,
                                      paddings,
                                      exclusive,
                                      data_format,
                                      pooling_type,
                                      global_pooling,
                                      adaptive,
                                      padding_algorithm,
                                      dx);
}

}  // namespace phi

using phi::dtype::float16;

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(pool2d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Pool2dGradGPUDNNKernel,
                   float,
                   float16) {}
PD_REGISTER_KERNEL(pool2d_double_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Pool2dDoubleGradGPUDNNKernel,
                   float,
                   float16) {}
PD_REGISTER_KERNEL(pool3d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Pool3dGradGPUDNNKernel,
                   float,
                   float16) {}
#else
PD_REGISTER_KERNEL(pool2d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Pool2dGradGPUDNNKernel,
                   float,
                   double,
                   float16) {}
PD_REGISTER_KERNEL(pool2d_double_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Pool2dDoubleGradGPUDNNKernel,
                   float,
                   double,
                   float16) {}
PD_REGISTER_KERNEL(pool3d_grad,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Pool3dGradGPUDNNKernel,
                   float,
                   double,
                   float16) {}
#endif
