#include "hip/hip_runtime.h"
// Copyright (c) 2023 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <map>
#include <unordered_map>
#include <vector>

#include "paddle/fluid/platform/bfloat16.h"
#include "paddle/fluid/platform/enforce.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/phi/kernels/gpudnn/mha_cudnn_frontend.h"

#define CUDNN_FRONTEND_UNUSED(X) ((void)X)

#ifdef PADDLE_WITH_CUDNN_FRONTEND

namespace phi {
namespace cudnn_fmha {

// Used for MHA
void generateMHAStrides(int64_t b,
                        int64_t h,
                        int64_t s_q,
                        int64_t s_kv,
                        int64_t d,
                        int64_t *strideA,
                        MHA_Layout layout,
                        MHA_Matrix matrix) {
  CUDNN_FRONTEND_UNUSED(b);
  constexpr int batch_dim_idx = 0;
  constexpr int head_dim_idx = 1;
  constexpr int seqlen_dim_idx = 2;
  constexpr int hidden_dim_idx = 3;

  constexpr int seqlen_transpose_dim_idx = 3;
  constexpr int hidden_transpose_dim_idx = 2;

  constexpr int seqlen_q_dim_idx = 2;
  constexpr int seqlen_kv_dim_idx = 3;

  switch (matrix) {
    case MHA_Matrix::Q_Matrix:
      if (layout == MHA_Layout::QKV_INTERLEAVED) {
        strideA[hidden_dim_idx] = 1;
        strideA[seqlen_dim_idx] = 3 * h * d;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_q * 3 * h * d;
      } else {
        strideA[hidden_dim_idx] = 1;
        strideA[seqlen_dim_idx] = h * d;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_q * h * d;
      }
      break;
    case MHA_Matrix::K_Matrix:
      if (layout == MHA_Layout::QKV_INTERLEAVED) {
        strideA[seqlen_transpose_dim_idx] = 3 * h * d;
        strideA[hidden_transpose_dim_idx] = 1;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_kv * 3 * h * d;
      } else if (layout == MHA_Layout::KV_INTERLEAVED) {
        strideA[seqlen_transpose_dim_idx] = 2 * h * d;
        strideA[hidden_transpose_dim_idx] = 1;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_kv * 2 * h * d;
      } else {
        strideA[seqlen_transpose_dim_idx] = h * d;
        strideA[hidden_transpose_dim_idx] = 1;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_kv * h * d;
      }
      break;
    case MHA_Matrix::V_Matrix:
      if (layout == MHA_Layout::QKV_INTERLEAVED) {
        strideA[hidden_dim_idx] = 1;
        strideA[seqlen_dim_idx] = 3 * h * d;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_kv * 3 * h * d;
      } else if (layout == MHA_Layout::KV_INTERLEAVED) {
        strideA[hidden_dim_idx] = 1;
        strideA[seqlen_dim_idx] = 2 * h * d;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_kv * 2 * h * d;
      } else {
        strideA[hidden_dim_idx] = 1;
        strideA[seqlen_dim_idx] = h * d;
        strideA[head_dim_idx] = d;
        strideA[batch_dim_idx] = s_kv * h * d;
      }
      break;
    case MHA_Matrix::S_Matrix:
      strideA[seqlen_kv_dim_idx] = 1;
      strideA[seqlen_q_dim_idx] = s_kv;
      strideA[head_dim_idx] = s_q * s_kv;
      strideA[batch_dim_idx] = h * s_q * s_kv;
      break;
    case MHA_Matrix::O_Matrix:
      strideA[seqlen_kv_dim_idx] = 1;
      strideA[seqlen_q_dim_idx] = h * d;
      strideA[head_dim_idx] = d;
      strideA[batch_dim_idx] = s_q * h * d;
      break;
  }
}

#define Q_ID 1
#define K_ID 2
#define V_ID 3
#define O_ID 4
#define S_ID 5
#define B_ID 6
#define D_CONST_ID 7
#define S_CONST_ID 8
#define Q_SEQLEN_ID 9
#define K_SEQLEN_ID 10
#define dQ_ID 11
#define dK_ID 12
#define dV_ID 13
#define dO_ID 14
#define MASK_VAL_ID 15
#define dS_ID 16

#define VIRTUAL_ID 20

static bool allowAllConfig(cudnnBackendDescriptor_t engine_config) {
  (void)engine_config;
  return false;
}

static cudnn_frontend::Tensor tensor_create(hipdnnDataType_t type,
                                            int64_t id,
                                            int64_t const *dim,
                                            int64_t const *stride,
                                            bool is_virtual,
                                            bool is_value) {
  int nbDims = 4;
  auto tensor_created =
      cudnn_frontend::TensorBuilder()
          .setDim(nbDims, dim)
          .setStride(nbDims, stride)
          .setId(id)
          .setAlignment(
              16)  // 16B alignment is needed to run a tensor core engine
          .setDataType(type)
          .setVirtual(is_virtual)
          .setByValue(is_value)
          .build();
  VLOG(10) << tensor_created.describe();
  return tensor_created;
}

static cudnn_frontend::PointWiseDesc pw_desc_create(hipdnnDataType_t type,
                                                    cudnnPointwiseMode_t mode) {
  auto pw_desc_created = cudnn_frontend::PointWiseDescBuilder()
                             .setMode(mode)
                             .setComputeType(type)
                             .build();

  VLOG(10) << pw_desc_created.describe();
  return pw_desc_created;
}

static cudnn_frontend::Operation unary_pw_op_create(
    cudnn_frontend::Tensor const &xDesc,
    cudnn_frontend::Tensor const &yDesc,
    cudnn_frontend::PointWiseDesc const &pwDesc) {
  auto pw_op_created = cudnn_frontend::OperationBuilder(
                           CUDNN_BACKEND_OPERATION_POINTWISE_DESCRIPTOR)
                           .setxDesc(xDesc)
                           .setyDesc(yDesc)
                           .setpwDesc(pwDesc)
                           .build();
  VLOG(10) << pw_op_created.describe();
  return pw_op_created;
}

static cudnn_frontend::Operation binary_pw_op_create(
    cudnn_frontend::Tensor const &xDesc,
    cudnn_frontend::Tensor const &bDesc,
    cudnn_frontend::Tensor const &yDesc,
    cudnn_frontend::PointWiseDesc const &pwDesc) {
  auto pw_op_created = cudnn_frontend::OperationBuilder(
                           CUDNN_BACKEND_OPERATION_POINTWISE_DESCRIPTOR)
                           .setxDesc(xDesc)
                           .setbDesc(bDesc)
                           .setyDesc(yDesc)
                           .setpwDesc(pwDesc)
                           .build();
  VLOG(10) << pw_op_created.describe();
  return pw_op_created;
}

static cudnn_frontend::Operation ternary_pw_op_create(
    cudnn_frontend::Tensor const &xDesc,
    cudnn_frontend::Tensor const &bDesc,
    cudnn_frontend::Tensor const &tDesc,
    cudnn_frontend::Tensor const &yDesc,
    cudnn_frontend::PointWiseDesc const &pwDesc) {
  auto pw_op_created = cudnn_frontend::OperationBuilder(
                           CUDNN_BACKEND_OPERATION_POINTWISE_DESCRIPTOR)
                           .setxDesc(xDesc)
                           .setbDesc(bDesc)
                           .settDesc(tDesc)
                           .setyDesc(yDesc)
                           .setpwDesc(pwDesc)
                           .build();
  VLOG(10) << pw_op_created.describe();
  return pw_op_created;
}

static void createScale(int64_t b,
                        int64_t h,
                        int64_t s_q,
                        int64_t s_kv,
                        int64_t d,
                        MHA_Layout layout,
                        hipdnnDataType_t tensorType,
                        std::vector<cudnn_frontend::Operation> &ops  // NOLINT
) {
  // scale
  int64_t scale_dim[4] = {1, 1, 1, 1};
  int64_t scale_stride[4] = {1, 1, 1, 1};

  int64_t k_dim[4] = {b, h, d, s_kv};
  int64_t k_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, k_stride, layout, MHA_Matrix::K_Matrix);

  auto scaleTensor = tensor_create(tensorType,
                                   S_CONST_ID,
                                   scale_dim,
                                   scale_stride,
                                   false,
                                   true);  // is by value
  auto kTensor = tensor_create(tensorType, K_ID, k_dim, k_stride, false, false);
  auto afterScaleKTensor = tensor_create(
      tensorType, VIRTUAL_ID, k_dim, k_stride, true, false);  // is virtual

  // Define the scale descriptor
  auto scaleDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

  // Create a Scale Node.
  auto scale_op =
      binary_pw_op_create(kTensor, scaleTensor, afterScaleKTensor, scaleDesc);

  ops.push_back(std::move(scale_op));
}

static cudnn_frontend::Tensor createBMM1(
    int64_t b,
    int64_t h,
    int64_t s_q,
    int64_t s_kv,
    int64_t d,
    MHA_Layout layout,
    hipdnnDataType_t tensorType,
    bool zero_s,
    std::vector<cudnn_frontend::Operation> &ops  // NOLINT
) {
  // Creates the necessary tensor descriptors
  int64_t q_dim[4] = {b, h, s_q, d};
  int64_t q_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, q_stride, layout, MHA_Matrix::Q_Matrix);

  int64_t k_dim[4] = {b, h, d, s_kv};
  int64_t k_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, k_stride, layout, MHA_Matrix::K_Matrix);

  int64_t p_dim[4] = {b, h, s_q, s_kv};
  int64_t p_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, p_stride, layout, MHA_Matrix::S_Matrix);

  int64_t seqlen_dim[4] = {b, 1, 1, 1};
  int64_t seqlen_stride[4] = {1, 1, 1, 1};

  auto qTensor = tensor_create(tensorType, Q_ID, q_dim, q_stride, false, false);
  auto afterScaleKTensor = tensor_create(
      tensorType, VIRTUAL_ID, k_dim, k_stride, true, false);  // is virtual
  // first GEMM output
  auto pTensor = tensor_create(HIPDNN_DATA_FLOAT,
                               VIRTUAL_ID + 1,
                               p_dim,
                               p_stride,
                               true,
                               false);  // is virtual

  auto seqlenQTensor = tensor_create(
      HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
  auto seqlenKTensor = tensor_create(
      HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);

  // Define the matmul 1 desc
  auto matmul_1_Desc = cudnn_frontend::MatMulDescBuilder()
                           .setComputeType(HIPDNN_DATA_FLOAT)
                           .build();
  if (zero_s) {
    auto matmul_1_Desc = cudnn_frontend::MatMulDescBuilder()
                             .setComputeType(HIPDNN_DATA_FLOAT)
                             .setPaddingValue(0.0f)
                             .build();
  }

  VLOG(10) << matmul_1_Desc.describe();

  // Create a matmul 1 Node
  auto matmul_op1 = cudnn_frontend::OperationBuilder(
                        CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                        .setaMatDesc(qTensor)
                        .setbMatDesc(afterScaleKTensor)
                        .setcMatDesc(pTensor)
                        .setmOverrideDesc(seqlenQTensor)
                        .setnOverrideDesc(seqlenKTensor)
                        .setmatmulDesc(matmul_1_Desc)
                        .build();

  VLOG(10) << matmul_op1.describe();

  ops.push_back(std::move(matmul_op1));

  return pTensor;
}

static cudnn_frontend::Tensor createBias(
    int64_t b,
    int64_t h,
    int64_t s_q,
    int64_t s_kv,
    int64_t d,
    MHA_Layout layout,
    hipdnnDataType_t tensorType,
    std::vector<cudnn_frontend::Operation> &ops,  // NOLINT
    cudnn_frontend::Tensor const &prevBlockOutputTensor) {
  cudnn_frontend::throw_if(ops.size() == 0,
                           "Bias op constructed incorrectly as the first one",
                           HIPDNN_STATUS_BAD_PARAM);

  int64_t b_dim[4] = {1, h, s_q, s_kv};
  int64_t b_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

  int64_t afterBias_dim[4] = {b, h, s_q, s_kv};
  int64_t afterBias_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, afterBias_stride, layout, MHA_Matrix::S_Matrix);

  // bias
  auto bTensor = tensor_create(tensorType, B_ID, b_dim, b_stride, false, false);
  // output
  auto afterBiasTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                       VIRTUAL_ID + 50,
                                       afterBias_dim,
                                       afterBias_stride,
                                       true,
                                       false);  // is virtual

  // Define the bias descriptor
  auto biasDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ADD);

  // Create a Bias Node.
  auto bias_op = binary_pw_op_create(
      prevBlockOutputTensor, bTensor, afterBiasTensor, biasDesc);

  ops.push_back(std::move(bias_op));

  return afterBiasTensor;
}

static cudnn_frontend::Tensor createMask(
    int64_t b,
    int64_t h,
    int64_t s_q,
    int64_t s_kv,
    int64_t d,
    MHA_Layout layout,
    bool is_causal_masking,
    hipdnnDataType_t tensorType,
    std::vector<cudnn_frontend::Operation> &ops,  // NOLINT
    cudnn_frontend::Tensor const &prevBlockOutputTensor,
    bool is_bprop) {
  CUDNN_FRONTEND_UNUSED(d);
  CUDNN_FRONTEND_UNUSED(layout);
  CUDNN_FRONTEND_UNUSED(tensorType);
  CUDNN_FRONTEND_UNUSED(is_bprop);

  cudnn_frontend::throw_if(
      ops.size() == 0,
      "Padding Mask constructed incorrectly as the first one",
      HIPDNN_STATUS_BAD_PARAM);

  // subtraction output
  int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
  int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

  int64_t seqlen_dim[4] = {b, 1, 1, 1};
  int64_t seqlen_stride[4] = {1, 1, 1, 1};

  int64_t maskVal_dim[4] = {1, 1, 1, 1};
  int64_t maskVal_stride[4] = {1, 1, 1, 1};

  // mask value to put in the masked pixels
  auto maskValTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                     MASK_VAL_ID,
                                     maskVal_dim,
                                     maskVal_stride,
                                     false,
                                     true);  // is by value

  auto seqlenQTensor = tensor_create(
      HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
  auto seqlenKTensor = tensor_create(
      HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
  // gen index row output
  auto rowIndexTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                      VIRTUAL_ID + 100,
                                      afterBMM1_dim,
                                      afterBMM1_stride,
                                      true,
                                      false);  // is virtual
  // gen index column output
  auto columnIndexTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                         VIRTUAL_ID + 101,
                                         afterBMM1_dim,
                                         afterBMM1_stride,
                                         true,
                                         false);  // is virtual
  // less than row output
  auto lessThanRowTensor = tensor_create(CUDNN_DATA_BOOLEAN,
                                         VIRTUAL_ID + 102,
                                         afterBMM1_dim,
                                         afterBMM1_stride,
                                         true,
                                         false);  // is virtual
                                                  // less than column output
  auto lessThanColTensor = tensor_create(CUDNN_DATA_BOOLEAN,
                                         VIRTUAL_ID + 103,
                                         afterBMM1_dim,
                                         afterBMM1_stride,
                                         true,
                                         false);  // is virtual
  // padding mask (lessthanRow && lessthanCol)
  auto paddingMaskTensor = tensor_create(CUDNN_DATA_BOOLEAN,
                                         VIRTUAL_ID + 104,
                                         afterBMM1_dim,
                                         afterBMM1_stride,
                                         true,
                                         false);  // is virtual
  // row >= col check for causal mask
  auto rowGreaterColTensor = tensor_create(CUDNN_DATA_BOOLEAN,
                                           VIRTUAL_ID + 105,
                                           afterBMM1_dim,
                                           afterBMM1_stride,
                                           true,
                                           false);  // is virtual
  // create causal mask (padding && row >= col)
  auto causalMaskTensor = tensor_create(CUDNN_DATA_BOOLEAN,
                                        VIRTUAL_ID + 106,
                                        afterBMM1_dim,
                                        afterBMM1_stride,
                                        true,
                                        false);  // is virtual

  // output after masking
  int64_t maskOutputTensor_id = VIRTUAL_ID + 107;
  int64_t maskOutputTensor_virtual = true;
  hipdnnDataType_t maskOutputTensor_dataType = HIPDNN_DATA_FLOAT;
  auto maskOutputTensor_reorderType = cudnn_frontend::
      cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_NONE;

  if (is_bprop) {
    maskOutputTensor_id = dS_ID;
    maskOutputTensor_virtual = false;
    maskOutputTensor_dataType = tensorType;
    maskOutputTensor_reorderType = cudnn_frontend::
        cudnnBackendTensorReordering_t::CUDNN_TENSOR_REORDERING_F16x16;
  }

  auto maskOutputTensor =
      cudnn_frontend::TensorBuilder()
          .setDim(4, afterBMM1_dim)
          .setStride(4, afterBMM1_stride)
          .setAlignment(
              16)  // 16B alignment is needed to run a tensor core engine
          .setByValue(false)
          .setDataType(maskOutputTensor_dataType)
          .setVirtual(maskOutputTensor_virtual)
          .setId(maskOutputTensor_id)
          .setReorderType(maskOutputTensor_reorderType)
          .build();

  // Define the gen index for row descriptor
  auto genIndexRowDesc = cudnn_frontend::PointWiseDescBuilder()
                             .setMode(CUDNN_POINTWISE_GEN_INDEX)
                             .setAxis(2)
                             .setComputeType(HIPDNN_DATA_FLOAT)
                             .build();
  VLOG(10) << genIndexRowDesc.describe();

  // Create a gen index Node.
  auto genIndexRow_op = unary_pw_op_create(
      prevBlockOutputTensor, rowIndexTensor, genIndexRowDesc);
  VLOG(10) << genIndexRow_op.describe();

  // Define the gen index for row descriptor
  auto genIndexColumnDesc = cudnn_frontend::PointWiseDescBuilder()
                                .setMode(CUDNN_POINTWISE_GEN_INDEX)
                                .setAxis(3)
                                .setComputeType(HIPDNN_DATA_FLOAT)
                                .build();
  VLOG(10) << genIndexColumnDesc.describe();

  // Create a gen index Node.
  auto genIndexColumn_op = unary_pw_op_create(
      prevBlockOutputTensor, columnIndexTensor, genIndexColumnDesc);

  // Define the less than comparison for row descriptor
  auto lessThanRowDesc =
      pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_CMP_LT);

  // Create a less than comparison for row Node.
  auto lessThanRow_op = binary_pw_op_create(
      rowIndexTensor, seqlenQTensor, lessThanRowTensor, lessThanRowDesc);

  // Define the less than comparison for column descriptor
  auto lessThanColDesc =
      pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_CMP_LT);

  // Create a less than comparison for col Node.
  auto lessThanCol_op = binary_pw_op_create(
      columnIndexTensor, seqlenKTensor, lessThanColTensor, lessThanColDesc);

  // Define the less than comparison for column descriptor
  auto paddingMaskAndDesc =
      pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_LOGICAL_AND);

  // Create a and node for combining lessThanRow and lessThanCol
  auto paddingMaskAnd_op = binary_pw_op_create(lessThanRowTensor,
                                               lessThanColTensor,
                                               paddingMaskTensor,
                                               paddingMaskAndDesc);

  // Define the greater than equal to comparison descriptor
  auto rowGreaterColDesc =
      pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_CMP_GE);

  // Create a greater than equal to Node.
  auto rowGreaterCol_op = binary_pw_op_create(rowIndexTensor,
                                              columnIndexTensor,
                                              rowGreaterColTensor,
                                              rowGreaterColDesc);

  // Define the and to create causal mask descriptor
  auto causalMaskAndDesc =
      pw_desc_create(CUDNN_DATA_BOOLEAN, CUDNN_POINTWISE_LOGICAL_AND);

  // Create a causal Mask Node.
  auto causalMaskAnd_op = binary_pw_op_create(paddingMaskTensor,
                                              rowGreaterColTensor,
                                              causalMaskTensor,
                                              causalMaskAndDesc);

  /////////////////// Apply the mask //////////////////////////

  auto maskTensor = (is_causal_masking) ? std::move(causalMaskTensor)
                                        : std::move(paddingMaskTensor);

  // Define the binary select to perform masking descriptor
  auto maskDesc =
      pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_BINARY_SELECT);

  // Create a binary select Node.
  auto mask_op = ternary_pw_op_create(prevBlockOutputTensor,
                                      maskValTensor,
                                      maskTensor,
                                      maskOutputTensor,
                                      maskDesc);

  ops.push_back(std::move(genIndexRow_op));
  ops.push_back(std::move(genIndexColumn_op));
  ops.push_back(std::move(lessThanRow_op));
  ops.push_back(std::move(lessThanCol_op));
  ops.push_back(std::move(paddingMaskAnd_op));
  if (is_causal_masking) ops.push_back(std::move(rowGreaterCol_op));
  if (is_causal_masking) ops.push_back(std::move(causalMaskAnd_op));
  ops.push_back(std::move(mask_op));

  return maskOutputTensor;
}

static cudnn_frontend::Tensor createSoftmaxForward(
    int64_t b,
    int64_t h,
    int64_t s_q,
    int64_t s_kv,
    int64_t d,
    MHA_Layout layout,
    bool enable_dropout,
    bool softmax_output_virtual,
    hipdnnDataType_t tensorType,
    std::vector<cudnn_frontend::Operation> &ops,  // NOLINT
    cudnn_frontend::Tensor const &prevBlockOutputTensor) {
  CUDNN_FRONTEND_UNUSED(d);
  CUDNN_FRONTEND_UNUSED(layout);

  int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
  int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

  int64_t afterReduction_dim[4] = {b, h, s_q, 1};
  int64_t afterReduction_stride[4] = {h * s_q, s_q, 1, 1};

  hipdnnDataType_t softmaxOutputType = (enable_dropout || softmax_output_virtual)
                                          ? HIPDNN_DATA_FLOAT
                                          : tensorType;
  uint64_t softmaxOutputName = softmax_output_virtual ? VIRTUAL_ID + 154 : S_ID;

  // max (x)
  auto afterMaxReductionTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                               VIRTUAL_ID + 150,
                                               afterReduction_dim,
                                               afterReduction_stride,
                                               true,
                                               false);  // is virtual
  // x - max(x)
  auto afterSubtractionTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                              VIRTUAL_ID + 151,
                                              afterBMM1_dim,
                                              afterBMM1_stride,
                                              true,
                                              false);  // is virtual
  // e^(x - max(x))
  auto afterExponentTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                           VIRTUAL_ID + 152,
                                           afterBMM1_dim,
                                           afterBMM1_stride,
                                           true,
                                           false);  // is virtual;
  // sum (e^(x - max(x)))
  auto afterAddReductionTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                               VIRTUAL_ID + 153,
                                               afterReduction_dim,
                                               afterReduction_stride,
                                               true,
                                               false);  // is virtual
  // divide (e/ sum(e))

  auto reorder_type = cudnn_frontend::cudnnBackendTensorReordering_t::
      CUDNN_TENSOR_REORDERING_F16x16;

  auto afterDivisionTensor =
      cudnn_frontend::TensorBuilder()
          .setDim(4, afterBMM1_dim)
          .setStride(4, afterBMM1_stride)
          .setId(softmaxOutputName)
          .setAlignment(
              16)  // 16B alignment is needed to run a tensor core engine
          .setDataType(softmaxOutputType)
          .setVirtual(softmax_output_virtual)
          .setByValue(false)
          .setReorderType(reorder_type)
          .build();

  // Define the reduction descriptor
  auto reductionMaxDesc = cudnn_frontend::ReductionDescBuilder()
                              .setComputeType(HIPDNN_DATA_FLOAT)
                              .setReductionOp(HIPDNN_REDUCE_TENSOR_MAX)
                              .build();
  VLOG(10) << reductionMaxDesc.describe();

  // Create a reduction max Node.
  auto reductionMax_op = cudnn_frontend::OperationBuilder(
                             CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                             .setxDesc(prevBlockOutputTensor)
                             .setyDesc(afterMaxReductionTensor)
                             .setreductionDesc(reductionMaxDesc)
                             .build();
  VLOG(10) << reductionMax_op.describe();

  // Define the subtract descriptor
  auto subtractDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_SUB);

  // Create a subtract Node.
  auto subtract_op = binary_pw_op_create(prevBlockOutputTensor,
                                         afterMaxReductionTensor,
                                         afterSubtractionTensor,
                                         subtractDesc);

  // Define the exponent descriptor
  auto exponentDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_EXP);

  // Create a exponent Node.
  auto exponent_op = unary_pw_op_create(
      afterSubtractionTensor, afterExponentTensor, exponentDesc);

  // Define the reduction descriptor
  auto reductionAddDesc = cudnn_frontend::ReductionDescBuilder()
                              .setComputeType(HIPDNN_DATA_FLOAT)
                              .setReductionOp(HIPDNN_REDUCE_TENSOR_ADD)
                              .build();
  VLOG(10) << reductionAddDesc.describe();

  // Create a reduction add Node.
  auto reductionAdd_op = cudnn_frontend::OperationBuilder(
                             CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                             .setxDesc(afterExponentTensor)
                             .setyDesc(afterAddReductionTensor)
                             .setreductionDesc(reductionAddDesc)
                             .build();

  VLOG(10) << reductionAdd_op.describe();

  // Define the division descriptor
  auto divisionDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_DIV);

  // Create a subtract Node.
  auto division_op = binary_pw_op_create(afterExponentTensor,
                                         afterAddReductionTensor,
                                         afterDivisionTensor,
                                         divisionDesc);

  ops.push_back(std::move(reductionMax_op));
  ops.push_back(std::move(subtract_op));
  ops.push_back(std::move(exponent_op));
  ops.push_back(std::move(reductionAdd_op));
  ops.push_back(std::move(division_op));

  return afterDivisionTensor;
}

static cudnn_frontend::Tensor createDropout(
    int64_t b,
    int64_t h,
    int64_t s_q,
    int64_t s_kv,
    int64_t d,
    int64_t seed,
    double probability,
    hipdnnDataType_t tensorType,
    // NOLINTNEXTLINE(runtime/references)
    std::vector<cudnn_frontend::Operation> &ops,
    cudnn_frontend::Tensor const &prevBlockOutputTensor) {
  CUDNN_FRONTEND_UNUSED(d);

  cudnn_frontend::throw_if(
      ops.size() == 0,
      "Dropout DAG constructed incorrectly as the first one",
      HIPDNN_STATUS_BAD_PARAM);

  int64_t afterBMM1_dim[4] = {b, h, s_q, s_kv};
  int64_t afterBMM1_stride[4] = {h * s_q * s_kv, s_q * s_kv, s_kv, 1};

  int64_t scale_dim[4] = {1, 1, 1, 1};
  int64_t scale_stride[4] = {1, 1, 1, 1};

  // mask for the dropout
  auto dropoutMaskTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                         VIRTUAL_ID + 200,
                                         afterBMM1_dim,
                                         afterBMM1_stride,
                                         true,
                                         false);  // is virtual

  auto reorder_type = cudnn_frontend::cudnnBackendTensorReordering_t::
      CUDNN_TENSOR_REORDERING_F16x16;

  // after dropout tensor
  auto afterDropoutTensor =
      cudnn_frontend::TensorBuilder()
          .setDim(4, afterBMM1_dim)
          .setStride(4, afterBMM1_stride)
          .setId(S_ID)
          .setAlignment(
              16)  // 16B alignment is needed to run a tensor core engine
          .setDataType(tensorType)
          .setVirtual(false)
          .setByValue(false)
          .setReorderType(reorder_type)
          .build();
  // scale after dropout
  auto scaleDropoutTensor = tensor_create(tensorType,
                                          D_CONST_ID,
                                          scale_dim,
                                          scale_stride,
                                          false,
                                          true);  // is by value
  // after Scale
  auto afterScaleTensor = tensor_create(tensorType,
                                        VIRTUAL_ID + 201,
                                        afterBMM1_dim,
                                        afterBMM1_stride,
                                        true,
                                        false);  // is virtual

  // Define the reduction descriptor
  auto rngDesc = cudnn_frontend::RngDescBuilder()
                     .setRngDistribution(CUDNN_RNG_DISTRIBUTION_BERNOULLI)
                     .setBernoulliDistProbability(1.0 - probability)
                     .build();
  VLOG(10) << rngDesc.describe();

  // Create a rng Node.
  auto rng_op =
      cudnn_frontend::OperationBuilder(CUDNN_BACKEND_OPERATION_RNG_DESCRIPTOR)
          .setyDesc(dropoutMaskTensor)
          .setSeed(seed)
          .setRngDesc(rngDesc)
          .build();

  VLOG(10) << rng_op.describe();

  // Define the multiply mask descriptor
  auto maskMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

  // Create a multiply mask Node.
  auto maskMul_op = binary_pw_op_create(prevBlockOutputTensor,
                                        dropoutMaskTensor,
                                        afterDropoutTensor,
                                        maskMulDesc);

  // Define the multiply scale descriptor
  auto scaleMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);

  // Create a multiply mask Node.
  auto scaleMul_op = binary_pw_op_create(
      afterDropoutTensor, scaleDropoutTensor, afterScaleTensor, scaleMulDesc);

  ops.push_back(std::move(rng_op));
  ops.push_back(std::move(maskMul_op));
  ops.push_back(std::move(scaleMul_op));

  return afterScaleTensor;
}

static void createBMM2(int64_t b,
                       int64_t h,
                       int64_t s_q,
                       int64_t s_kv,
                       int64_t d,
                       MHA_Layout layout,
                       hipdnnDataType_t tensorType,
                       std::vector<cudnn_frontend::Operation> &ops,  // NOLINT
                       cudnn_frontend::Tensor const &prevBlockOutputTensor) {
  cudnn_frontend::throw_if(ops.size() == 0,
                           "BMM2 op constructed incorrectly as the first one",
                           HIPDNN_STATUS_BAD_PARAM);

  int64_t seqlen_dim[4] = {b, 1, 1, 1};
  int64_t seqlen_stride[4] = {1, 1, 1, 1};

  int64_t v_dim[4] = {b, h, s_kv, d};
  int64_t v_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, v_stride, layout, MHA_Matrix::V_Matrix);

  int64_t o_dim[4] = {b, h, s_q, d};
  int64_t o_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, o_stride, layout, MHA_Matrix::O_Matrix);

  auto seqlenQTensor = tensor_create(
      HIPDNN_DATA_INT32, Q_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
  auto seqlenKTensor = tensor_create(
      HIPDNN_DATA_INT32, K_SEQLEN_ID, seqlen_dim, seqlen_stride, false, false);
  auto vTensor = tensor_create(tensorType, V_ID, v_dim, v_stride, false, false);
  // second GEMM output
  auto oTensor = tensor_create(tensorType, O_ID, o_dim, o_stride, false, false);

  // Define the matmul 2 desc
  auto matmul_2_Desc = cudnn_frontend::MatMulDescBuilder()
                           .setComputeType(HIPDNN_DATA_FLOAT)
                           .setPaddingValue(0.0f)
                           .build();
  VLOG(10) << matmul_2_Desc.describe();

  // Create a matmul 2 Node
  auto matmul_op2 = cudnn_frontend::OperationBuilder(
                        CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                        .setaMatDesc(prevBlockOutputTensor)
                        .setbMatDesc(vTensor)
                        .setcMatDesc(oTensor)
                        .setmOverrideDesc(seqlenQTensor)
                        .setkOverrideDesc(seqlenKTensor)
                        .setmatmulDesc(matmul_2_Desc)
                        .build();

  VLOG(10) << matmul_op2.describe();

  ops.push_back(std::move(matmul_op2));
}

static cudnn_frontend::Tensor createSoftmaxBackward(
    int64_t b,
    int64_t h,
    int64_t s_q,
    int64_t s_kv,
    int64_t d,
    MHA_Layout layout,
    hipdnnDataType_t tensorType,
    // NOLINTNEXTLINE(runtime/references)
    std::vector<cudnn_frontend::Operation> &ops,
    cudnn_frontend::Tensor const &yTensor,
    cudnn_frontend::Tensor const &dyTensor) {
  CUDNN_FRONTEND_UNUSED(tensorType);

  cudnn_frontend::throw_if(
      ops.size() == 0,
      "Softmax backward constructed incorrectly as the first one",
      HIPDNN_STATUS_BAD_PARAM);

  int64_t p_dim[4] = {b, h, s_q, s_kv};
  int64_t p_stride[4];
  generateMHAStrides(
      b, h, s_q, s_kv, d, p_stride, layout, MHA_Matrix::S_Matrix);

  int64_t p_reduction_dim[4] = {b, h, s_q, 1};
  int64_t p_reduction_stride[4];

  p_reduction_stride[3] = 1;
  p_reduction_stride[2] = 1;
  p_reduction_stride[1] = s_q;
  p_reduction_stride[0] = s_q * h;

  int64_t const_dim[4] = {1, 1, 1, 1};
  int64_t const_stride[4] = {1, 1, 1, 1};

  // creating all tensors
  auto softmaxScaleTensor = tensor_create(
      HIPDNN_DATA_FLOAT, S_CONST_ID, const_dim, const_stride, false, true);
  auto dyMulYTensor = tensor_create(
      HIPDNN_DATA_FLOAT, VIRTUAL_ID + 250, p_dim, p_stride, true, false);
  auto dxAfterReductionTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                              VIRTUAL_ID + 251,
                                              p_reduction_dim,
                                              p_reduction_stride,
                                              true,
                                              false);
  auto dxAfterSubtractionTensor = tensor_create(
      HIPDNN_DATA_FLOAT, VIRTUAL_ID + 252, p_dim, p_stride, true, false);
  auto dxUnscaleTensor = tensor_create(
      HIPDNN_DATA_FLOAT, VIRTUAL_ID + 253, p_dim, p_stride, true, false);
  auto dxTensor = tensor_create(
      HIPDNN_DATA_FLOAT, VIRTUAL_ID + 254, p_dim, p_stride, true, false);

  // creating all ops
  // mul (y * dy)
  auto mul_1_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
  auto mul_1_op =
      binary_pw_op_create(yTensor, dyTensor, dyMulYTensor, mul_1_desc);

  // reduction add sum (y * dy)
  auto reductionAddDesc = cudnn_frontend::ReductionDescBuilder()
                              .setComputeType(HIPDNN_DATA_FLOAT)
                              .setReductionOp(HIPDNN_REDUCE_TENSOR_ADD)
                              .build();
  VLOG(10) << reductionAddDesc.describe();

  auto reductionAdd_op = cudnn_frontend::OperationBuilder(
                             CUDNN_BACKEND_OPERATION_REDUCTION_DESCRIPTOR)
                             .setxDesc(dyMulYTensor)
                             .setyDesc(dxAfterReductionTensor)
                             .setreductionDesc(reductionAddDesc)
                             .build();

  VLOG(10) << reductionAdd_op.describe();

  // subtraction (dy - sum(y * dy))
  auto sub_0_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_SUB);
  auto sub_0_op = binary_pw_op_create(
      dyTensor, dxAfterReductionTensor, dxAfterSubtractionTensor, sub_0_desc);

  // mul (y * (dy - sum(y * dy)))
  auto mul_2_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
  auto mul_2_op = binary_pw_op_create(
      yTensor, dxAfterSubtractionTensor, dxUnscaleTensor, mul_2_desc);

  // mul (scale * dx)
  auto mul_3_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
  auto mul_3_op = binary_pw_op_create(
      dxUnscaleTensor, softmaxScaleTensor, dxTensor, mul_3_desc);

  ops.push_back(std::move(mul_1_op));
  ops.push_back(std::move(reductionAdd_op));
  ops.push_back(std::move(sub_0_op));
  ops.push_back(std::move(mul_2_op));
  ops.push_back(std::move(mul_3_op));

  return dxTensor;
}

struct FMHADescriptor {
  std::int64_t b;
  std::int64_t h;
  std::int64_t s_q;
  std::int64_t s_kv;
  std::int64_t d;
  std::int64_t seed;
  float scaling_factor;
  float dropout_probability;
  bool is_causal_masking;
  MHA_Layout layout;
  MHA_Bias_Type bias_type;
  hipdnnDataType_t tensor_type;

  bool operator<(const FMHADescriptor &rhs) const {
    return std::tie(b,
                    h,
                    s_q,
                    s_kv,
                    d,
                    seed,
                    scaling_factor,
                    dropout_probability,
                    is_causal_masking,
                    layout,
                    bias_type,
                    tensor_type) < std::tie(rhs.b,
                                            rhs.h,
                                            rhs.s_q,
                                            rhs.s_kv,
                                            rhs.d,
                                            rhs.seed,
                                            rhs.scaling_factor,
                                            rhs.dropout_probability,
                                            rhs.is_causal_masking,
                                            rhs.layout,
                                            rhs.bias_type,
                                            rhs.tensor_type);
  }
};
}  // namespace cudnn_fmha
}  // namespace phi

using namespace phi::cudnn_fmha;  // NOLINT

void run_cudnn_fmha_fwd(int64_t b,
                        int64_t h,
                        int64_t s_q,
                        int64_t s_kv,
                        int64_t d,
                        int64_t seed,
                        MHA_Layout layout,
                        float scaling_factor,
                        double dropout_probability,
                        MHA_Bias_Type bias_type,
                        bool is_causal_masking,
                        void *devPtrQ,
                        void *devPtrK,
                        void *devPtrV,
                        void *devPtrS,
                        void *devPtrO,
                        void *devPtrBias,
                        void *devActualSeqlenQ,
                        void *devActualSeqlenKV,
                        hipdnnDataType_t tensorType,
                        hipdnnHandle_t handle_) {
  // hipdnnHandle_t handle_;
  try {
    // Create cudnn handle
    // checkCudnnErr(hipdnnCreate(&handle_));

    FMHADescriptor descriptor{b,
                              h,
                              s_q,
                              s_kv,
                              d,
                              seed,
                              scaling_factor,
                              static_cast<float>(dropout_probability),
                              is_causal_masking,
                              layout,
                              bias_type,
                              tensorType};

    using CacheType = std::map<FMHADescriptor, cudnn_frontend::ExecutionPlan>;
    static CacheType fmha_fprop_cache;

    bool enable_dropout = (dropout_probability != 0.0f);

    // Get plan from cache if cache is available, otherwise create one
    auto get_plan = [&](CacheType &cache, const FMHADescriptor &descriptor) {
      // if hit, return
      auto it = cache.find(descriptor);
      if (it != cache.end()) {
        auto plan = it->second;
        return plan;
      }

      // otherwise, build the op_graph and the plan. Then update cache
      std::vector<cudnn_frontend::Operation const *> all_ops;
      std::vector<cudnn_frontend::Operation> ops;

      createScale(b, h, s_q, s_kv, d, layout, tensorType, ops);

      auto zero_s = (bias_type != MHA_Bias_Type::NO_BIAS);
      auto bmm1_output =
          createBMM1(b, h, s_q, s_kv, d, layout, tensorType, zero_s, ops);

      if (bias_type != MHA_Bias_Type::NO_BIAS) {
        createBias(b, h, s_q, s_kv, d, layout, tensorType, ops, bmm1_output);
      }

      auto mask_output = createMask(b,
                                    h,
                                    s_q,
                                    s_kv,
                                    d,
                                    layout,
                                    is_causal_masking,
                                    tensorType,
                                    ops,
                                    bmm1_output,
                                    false);

      cudnn_frontend::throw_if(dropout_probability == 1.0f,
                               "Dropout probability cannot be 1.0",
                               HIPDNN_STATUS_BAD_PARAM);

      bool softmax_output_virtual = enable_dropout || devPtrS == nullptr;
      auto softmax_output = createSoftmaxForward(b,
                                                 h,
                                                 s_q,
                                                 s_kv,
                                                 d,
                                                 layout,
                                                 enable_dropout,
                                                 softmax_output_virtual,
                                                 tensorType,
                                                 ops,
                                                 mask_output);

      if (dropout_probability != 0.0f) {
        auto dropout_output = createDropout(b,
                                            h,
                                            s_q,
                                            s_kv,
                                            d,
                                            seed,
                                            dropout_probability,
                                            tensorType,
                                            ops,
                                            softmax_output);
        createBMM2(b, h, s_q, s_kv, d, layout, tensorType, ops, dropout_output);
      } else {
        createBMM2(b, h, s_q, s_kv, d, layout, tensorType, ops, softmax_output);
      }

      for (unsigned int i = 0; i < ops.size(); i++) {
        all_ops.push_back(&ops[i]);
      }
      VLOG(10) << "Total ops created: " << ops.size();

      // Create an Operation Graph
      auto opGraph = cudnn_frontend::OperationGraphBuilder()
                         .setHandle(handle_)
                         .setOperationGraph(all_ops.size(), all_ops.data())
                         .build();

      cudnn_frontend::EngineConfigList filtered_configs;
      auto statuses =
          cudnn_frontend::get_heuristics_list<1>({"heuristics_instant"},
                                                 opGraph,
                                                 allowAllConfig,
                                                 filtered_configs,
                                                 true);

      if (filtered_configs.size() == 0) {
        cudnn_frontend::set_error_and_throw_exception(
            nullptr,
            HIPDNN_STATUS_NOT_SUPPORTED,
            "run_mha_fprop: No config returned by the heuristics");
      }
      auto plan = cudnn_frontend::ExecutionPlanBuilder()
                      .setHandle(handle_)
                      .setEngineConfig(filtered_configs[0], opGraph.getTag())
                      .build();
      cache.insert({descriptor, plan});
      return plan;
    };

    auto plan = get_plan(fmha_fprop_cache, descriptor);
    VLOG(10) << "Plan tag: " << plan.getTag();

    auto workspace_size = plan.getWorkspaceSize();
    VLOG(10) << plan.describe() << " requires workspace " << workspace_size;

    void *workspace_ptr = nullptr;
    if (workspace_size > 0) {
      PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(&workspace_ptr, workspace_size));
    }

    std::set<std::pair<uint64_t, void *>> data_ptrs;
    // change this if you have access to float_min
    float negInfinity = -1.0E+20;
    auto scale_dropout = paddle::platform::float16(
                             static_cast<float>(1 / (1 - dropout_probability)))
                             .x;

    // add all the data pointers to be used in the variant pack
    data_ptrs.insert(std::pair<uint64_t, void *>(Q_ID, devPtrQ));
    data_ptrs.insert(std::pair<uint64_t, void *>(K_ID, devPtrK));
    data_ptrs.insert(std::pair<uint64_t, void *>(V_ID, devPtrV));
    data_ptrs.insert(
        std::pair<uint64_t, void *>(Q_SEQLEN_ID, devActualSeqlenQ));
    data_ptrs.insert(
        std::pair<uint64_t, void *>(K_SEQLEN_ID, devActualSeqlenKV));
    data_ptrs.insert(std::pair<uint64_t, void *>(MASK_VAL_ID, &negInfinity));

    if (tensorType == HIPDNN_DATA_FLOAT) {
      data_ptrs.insert(
          std::pair<uint64_t, void *>(S_CONST_ID, &scaling_factor));
    } else if (tensorType == HIPDNN_DATA_HALF) {
      auto scaling_factor_fp16 = paddle::platform::float16(scaling_factor).x;
      data_ptrs.insert(
          std::pair<uint64_t, void *>(S_CONST_ID, &scaling_factor_fp16));
    } else if (tensorType == CUDNN_DATA_BFLOAT16) {
      auto scaling_factor_bf16 = paddle::platform::bfloat16(scaling_factor).x;
      data_ptrs.insert(
          std::pair<uint64_t, void *>(S_CONST_ID, &scaling_factor_bf16));
    } else {
      std::cerr << "Not supported tensorType." << std::endl;
    }

    data_ptrs.insert(std::pair<uint64_t, void *>(O_ID, devPtrO));

    if (bias_type != MHA_Bias_Type::NO_BIAS) {
      data_ptrs.insert(std::pair<uint64_t, void *>(B_ID, devPtrBias));
    }

    if (devPtrS != nullptr) {
      data_ptrs.insert(std::pair<uint64_t, void *>(S_ID, devPtrS));
    }

    if (enable_dropout) {
      data_ptrs.insert(std::pair<uint64_t, void *>(D_CONST_ID, &scale_dropout));
    }

    auto variantPack = cudnn_frontend::VariantPackBuilder()
                           .setWorkspacePointer(workspace_ptr)
                           .setDataPointers(data_ptrs)
                           .build();
    VLOG(10) << "variantPack " << variantPack.describe();
    hipdnnStatus_t status = phi::dynload::cudnnBackendExecute(
        handle_, plan.get_raw_desc(), variantPack.get_raw_desc());
    if (workspace_size > 0) {
      PADDLE_ENFORCE_GPU_SUCCESS(hipFree(workspace_ptr));
    }

    // checkCudnnErr(hipdnnDestroy(handle_));

    cudnn_frontend::throw_if(
        [status]() { return (status != HIPDNN_STATUS_SUCCESS); },
        "Plan execute error",
        status);
  } catch (cudnn_frontend::cudnnException &e) {
    struct hipDeviceProp_t prop;
    PADDLE_ENFORCE_GPU_SUCCESS(hipGetDeviceProperties(&prop, 0));

    // this example is only for GA100 cards (cudnn Version >= 8700) and GH100
    // cards (cudnn Version >= 8800)
    if (!((prop.major == 8 && prop.minor == 0) ||
          (prop.major == 9 && prop.minor == 0 && CUDNN_VERSION >= 8800)) &&
        (e.getCudnnStatus() == HIPDNN_STATUS_ARCH_MISMATCH ||
         e.getCudnnStatus() == HIPDNN_STATUS_NOT_SUPPORTED)) {
      std::cout << "Only supported for GA100 (cuDNN >= 8700) and "
                   "GH100 (cuDNN >= 8800) GPUs"
                << std::endl;
    } else {
      std::cout << "[ERROR] Exception " << e.what() << std::endl;
      // CHECK(false);
    }
  }
}

void run_cudnn_fmha_bwd(int64_t b,
                        int64_t h,
                        int64_t s_q,
                        int64_t s_kv,
                        int64_t d,
                        MHA_Layout layout,
                        float scaling_factor,
                        float dropout_probability,
                        bool is_causal_masking,
                        void *devPtrQ,
                        void *devPtrK,
                        void *devPtrV,
                        void *devPtrS,
                        void *devPtrdQ,
                        void *devPtrdK,
                        void *devPtrdV,
                        void *devPtrdO,
                        void *devPtrdS,
                        void *devActualSeqlenQ,
                        void *devActualSeqlenKV,
                        hipdnnDataType_t tensorType,
                        hipdnnHandle_t handle_) {
  // hipdnnHandle_t handle_;
  try {
    // Create cudnn handle
    // checkCudnnErr(hipdnnCreate(&handle_));

    FMHADescriptor descriptor{b,
                              h,
                              s_q,
                              s_kv,
                              d,
                              0,
                              scaling_factor,
                              static_cast<float>(dropout_probability),
                              is_causal_masking,
                              layout,
                              MHA_Bias_Type::NO_BIAS,
                              tensorType};

    using CacheType = std::map<FMHADescriptor, cudnn_frontend::ExecutionPlan>;
    static CacheType fmha_bprop_cache;

    auto get_plan = [&](CacheType &cache, const FMHADescriptor &descriptor) {
      auto it = cache.find(descriptor);
      if (it != cache.end()) {
        return it->second;
      }

      std::vector<cudnn_frontend::Operation const *> all_ops;
      std::vector<cudnn_frontend::Operation> ops;

      // Creates the necessary tensor descriptors
      int64_t q_dim[4] = {b, h, s_q, d};
      int64_t q_stride[4];
      generateMHAStrides(
          b, h, s_q, s_kv, d, q_stride, layout, MHA_Matrix::Q_Matrix);

      int64_t k_dim[4] = {b, h, s_kv, d};
      int64_t k_stride[4];
      generateMHAStrides(
          b,
          h,
          s_q,
          s_kv,
          d,
          k_stride,
          layout,
          MHA_Matrix::V_Matrix);  // type is correct as K is not transposed

      int64_t v_dim[4] = {b, h, d, s_kv};
      int64_t v_stride[4];
      generateMHAStrides(
          b,
          h,
          s_q,
          s_kv,
          d,
          v_stride,
          layout,
          MHA_Matrix::K_Matrix);  // type is correct as V is transposed

      int64_t p_dim[4] = {b, h, s_q, s_kv};
      int64_t p_stride[4];
      generateMHAStrides(
          b, h, s_q, s_kv, d, p_stride, layout, MHA_Matrix::S_Matrix);

      int64_t p_transpose_dim[4] = {b, h, s_kv, s_q};
      int64_t p_transpose_stride[4];
      p_transpose_stride[0] = p_stride[0];
      p_transpose_stride[1] = p_stride[1];
      p_transpose_stride[2] = p_stride[3];
      p_transpose_stride[3] = p_stride[2];

      int64_t o_dim[4] = {b, h, s_q, d};
      int64_t o_stride[4];
      generateMHAStrides(
          b, h, s_q, s_kv, d, o_stride, layout, MHA_Matrix::O_Matrix);

      int64_t seqlen_dim[4] = {b, 1, 1, 1};
      int64_t seqlen_stride[4] = {1, 1, 1, 1};

      int64_t scale_dim[4] = {1, 1, 1, 1};
      int64_t scale_stride[4] = {1, 1, 1, 1};

      // inputs to fprop
      auto qTensor =
          tensor_create(tensorType, Q_ID, q_dim, q_stride, false, false);
      auto kTensor =
          tensor_create(tensorType, K_ID, k_dim, k_stride, false, false);
      auto vTensor =
          tensor_create(tensorType, V_ID, v_dim, v_stride, false, false);
      auto seqlenQTensor = tensor_create(HIPDNN_DATA_INT32,
                                         Q_SEQLEN_ID,
                                         seqlen_dim,
                                         seqlen_stride,
                                         false,
                                         false);
      auto seqlenKTensor = tensor_create(HIPDNN_DATA_INT32,
                                         K_SEQLEN_ID,
                                         seqlen_dim,
                                         seqlen_stride,
                                         false,
                                         false);

      // gradient of the output
      auto doTensor =
          tensor_create(tensorType, dO_ID, o_dim, o_stride, false, false);

      auto reorder_type = cudnn_frontend::cudnnBackendTensorReordering_t::
          CUDNN_TENSOR_REORDERING_F16x16;

      // activation from fprop
      auto pTensor =
          cudnn_frontend::TensorBuilder()
              .setDim(4, p_dim)
              .setStride(4, p_stride)
              .setId(S_ID)
              .setAlignment(
                  16)  // 16B alignment is needed to run a tensor core engine
              .setDataType(tensorType)
              .setVirtual(false)
              .setByValue(false)
              .setReorderType(reorder_type)
              .build();

      // outputs from bprop
      auto dqTensor =
          tensor_create(tensorType, dQ_ID, q_dim, q_stride, false, false);
      auto dkTensor =
          tensor_create(tensorType, dK_ID, k_dim, k_stride, false, false);
      auto dvTensor =
          tensor_create(tensorType,
                        dV_ID,
                        k_dim,
                        k_stride,
                        false,
                        false);  // not transposed therefore k_dim and k_stride

      ////////////////////////////////////////////////////////
      // start creating the ops and the intermediate tensors
      auto pReshapeTensor = tensor_create(tensorType,
                                          VIRTUAL_ID + 300,
                                          p_transpose_dim,
                                          p_transpose_stride,
                                          true,
                                          false);

      // reshape to perform transpose and make pReshape
      auto reshape_op = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                            .setxDesc(pTensor)
                            .setyDesc(pReshapeTensor)
                            .build();

      VLOG(10) << reshape_op.describe();
      ops.push_back(std::move(reshape_op));

      // scale dropout
      auto dropoutScaleTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                              D_CONST_ID,
                                              scale_dim,
                                              scale_stride,
                                              false,
                                              true);  // is by value
      auto pAfterScaleTensor = tensor_create(tensorType,
                                             VIRTUAL_ID + 301,
                                             p_transpose_dim,
                                             p_transpose_stride,
                                             true,
                                             false);

      auto scaleMulDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
      auto scaleMul_op = binary_pw_op_create(
          pReshapeTensor, dropoutScaleTensor, pAfterScaleTensor, scaleMulDesc);
      ops.push_back(std::move(scaleMul_op));

      // perform absolute operation to remove the mask bit
      auto pTransposeAfterAbsTensor = tensor_create(tensorType,
                                                    VIRTUAL_ID + 302,
                                                    p_transpose_dim,
                                                    p_transpose_stride,
                                                    true,
                                                    false);

      auto absDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ABS);
      auto abs_op = unary_pw_op_create(
          pAfterScaleTensor, pTransposeAfterAbsTensor, absDesc);
      ops.push_back(std::move(abs_op));

      // matmul to calculate dvTensor
      auto matmul_0_Desc = cudnn_frontend::MatMulDescBuilder()
                               .setComputeType(HIPDNN_DATA_FLOAT)
                               .setPaddingValue(0.0f)
                               .build();
      VLOG(10) << matmul_0_Desc.describe();

      auto matmul_op0 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(pTransposeAfterAbsTensor)
                            .setbMatDesc(doTensor)
                            .setcMatDesc(dvTensor)
                            .setmOverrideDesc(seqlenKTensor)
                            .setkOverrideDesc(seqlenQTensor)
                            .setmatmulDesc(matmul_0_Desc)
                            .build();

      VLOG(10) << matmul_op0.describe();

      ops.push_back(std::move(matmul_op0));

      // matmul to calculate dpTensor
      auto dpTensor = tensor_create(
          HIPDNN_DATA_FLOAT, VIRTUAL_ID + 303, p_dim, p_stride, true, false);

      auto matmul_1_Desc = cudnn_frontend::MatMulDescBuilder()
                               .setComputeType(HIPDNN_DATA_FLOAT)
                               .build();
      VLOG(10) << matmul_1_Desc.describe();

      auto matmul_op1 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(doTensor)
                            .setbMatDesc(vTensor)
                            .setcMatDesc(dpTensor)
                            .setmOverrideDesc(seqlenQTensor)
                            .setnOverrideDesc(seqlenKTensor)
                            .setmatmulDesc(matmul_1_Desc)
                            .build();

      VLOG(10) << matmul_op1.describe();

      ops.push_back(std::move(matmul_op1));

      // mask the values which were dropped in dropout
      auto pAbsTensor = tensor_create(
          tensorType, VIRTUAL_ID + 304, p_dim, p_stride, true, false);

      auto p_absDesc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_ABS);
      auto p_abs_op = unary_pw_op_create(pTensor, pAbsTensor, p_absDesc);
      ops.push_back(std::move(p_abs_op));

      // create the dropout mask
      auto zeroTensor = tensor_create(HIPDNN_DATA_FLOAT,
                                      MASK_VAL_ID,
                                      scale_dim,
                                      scale_stride,
                                      false,
                                      true);  // is by value
      auto dropoutMaskTensor = tensor_create(
          CUDNN_DATA_BOOLEAN, VIRTUAL_ID + 305, p_dim, p_stride, true, false);

      auto greater_than_0_desc =
          pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_CMP_GT);
      auto greater_than_0_op = binary_pw_op_create(
          pTensor, zeroTensor, dropoutMaskTensor, greater_than_0_desc);
      ops.push_back(std::move(greater_than_0_op));

      // scale for the dropout
      auto dpAfterScaleTensor = tensor_create(
          HIPDNN_DATA_FLOAT, VIRTUAL_ID + 306, p_dim, p_stride, true, false);

      auto mul_0_desc = pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_MUL);
      auto mul_0_op = binary_pw_op_create(
          dpTensor, dropoutScaleTensor, dpAfterScaleTensor, mul_0_desc);
      ops.push_back(std::move(mul_0_op));

      // drop the values based on the dropout mask
      auto dpAfterDropoutTensor = tensor_create(
          HIPDNN_DATA_FLOAT, VIRTUAL_ID + 307, p_dim, p_stride, true, false);

      auto selection_0_desc =
          pw_desc_create(HIPDNN_DATA_FLOAT, CUDNN_POINTWISE_BINARY_SELECT);
      auto selection_0_op = ternary_pw_op_create(dpAfterScaleTensor,
                                                 zeroTensor,
                                                 dropoutMaskTensor,
                                                 dpAfterDropoutTensor,
                                                 selection_0_desc);
      ops.push_back(std::move(selection_0_op));

      // softmax backward
      auto dsTensor = createSoftmaxBackward(b,
                                            h,
                                            s_q,
                                            s_kv,
                                            d,
                                            layout,
                                            tensorType,
                                            ops,
                                            pAbsTensor,
                                            dpAfterDropoutTensor);

      // mask
      auto dsAfterMaskTensor = createMask(b,
                                          h,
                                          s_q,
                                          s_kv,
                                          d,
                                          layout,
                                          is_causal_masking,
                                          tensorType,
                                          ops,
                                          dsTensor,
                                          true);

      // matmul to calculate dqTensor
      auto matmul_2_Desc = cudnn_frontend::MatMulDescBuilder()
                               .setComputeType(HIPDNN_DATA_FLOAT)
                               .setPaddingValue(0.0f)
                               .build();
      VLOG(10) << matmul_2_Desc.describe();

      auto matmul_op2 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(dsAfterMaskTensor)
                            .setbMatDesc(kTensor)
                            .setcMatDesc(dqTensor)
                            .setmOverrideDesc(seqlenQTensor)
                            .setkOverrideDesc(seqlenKTensor)
                            .setmatmulDesc(matmul_2_Desc)
                            .build();

      VLOG(10) << matmul_op2.describe();

      ops.push_back(std::move(matmul_op2));

      // reshape for transpose of ds
      auto dsAfterMaskReshapeTensor = tensor_create(tensorType,
                                                    VIRTUAL_ID + 308,
                                                    p_transpose_dim,
                                                    p_transpose_stride,
                                                    true,
                                                    false);

      auto reshape_2_op = cudnn_frontend::OperationBuilder(
                              CUDNN_BACKEND_OPERATION_RESHAPE_DESCRIPTOR)
                              .setxDesc(dsAfterMaskTensor)
                              .setyDesc(dsAfterMaskReshapeTensor)
                              .build();

      VLOG(10) << reshape_2_op.describe();
      ops.push_back(std::move(reshape_2_op));

      // matmul to calculate dkTensor
      auto matmul_3_Desc = cudnn_frontend::MatMulDescBuilder()
                               .setComputeType(HIPDNN_DATA_FLOAT)
                               .setPaddingValue(0.0f)
                               .build();
      VLOG(10) << matmul_3_Desc.describe();

      auto matmul_op3 = cudnn_frontend::OperationBuilder(
                            CUDNN_BACKEND_OPERATION_MATMUL_DESCRIPTOR)
                            .setaMatDesc(dsAfterMaskReshapeTensor)
                            .setbMatDesc(qTensor)
                            .setcMatDesc(dkTensor)
                            .setmOverrideDesc(seqlenKTensor)
                            .setkOverrideDesc(seqlenQTensor)
                            .setmatmulDesc(matmul_3_Desc)
                            .build();

      VLOG(10) << matmul_op3.describe();

      ops.push_back(std::move(matmul_op3));

      /////////////////////////////////////////////////////////////////

      VLOG(10) << "Total ops created: " << ops.size();

      for (unsigned int i = 0; i < ops.size(); i++) {
        all_ops.push_back(&ops[i]);
      }

      // Create an Operation Graph
      auto opGraph = cudnn_frontend::OperationGraphBuilder()
                         .setHandle(handle_)
                         .setOperationGraph(all_ops.size(), all_ops.data())
                         .build();

      cudnn_frontend::EngineConfigList filtered_configs;
      auto statuses =
          cudnn_frontend::get_heuristics_list<1>({"heuristics_instant"},
                                                 opGraph,
                                                 allowAllConfig,
                                                 filtered_configs,
                                                 true);

      if (filtered_configs.size() == 0) {
        cudnn_frontend::set_error_and_throw_exception(
            nullptr,
            HIPDNN_STATUS_NOT_SUPPORTED,
            "run_mha_bprop: No config returned by the heuristics");
      }

      auto plan = cudnn_frontend::ExecutionPlanBuilder()
                      .setHandle(handle_)
                      .setEngineConfig(filtered_configs[0], opGraph.getTag())
                      .build();
      cache.insert({descriptor, plan});
      return plan;
    };

    auto plan = get_plan(fmha_bprop_cache, descriptor);
    VLOG(10) << "Plan tag: " << plan.getTag();

    auto workspace_size = plan.getWorkspaceSize();
    VLOG(10) << plan.describe() << " requires workspace " << workspace_size;

    void *workspace_ptr = nullptr;
    if (workspace_size > 0) {
      PADDLE_ENFORCE_GPU_SUCCESS(hipMalloc(&workspace_ptr, workspace_size));
    }

    std::set<std::pair<uint64_t, void *>> data_ptrs;
    // add all the data pointers to be used in the variant pack
    data_ptrs.insert(std::pair<uint64_t, void *>(dQ_ID, devPtrdQ));
    data_ptrs.insert(std::pair<uint64_t, void *>(dK_ID, devPtrdK));
    data_ptrs.insert(std::pair<uint64_t, void *>(dV_ID, devPtrdV));

    data_ptrs.insert(std::pair<uint64_t, void *>(Q_ID, devPtrQ));
    data_ptrs.insert(std::pair<uint64_t, void *>(K_ID, devPtrK));
    data_ptrs.insert(std::pair<uint64_t, void *>(V_ID, devPtrV));
    data_ptrs.insert(std::pair<uint64_t, void *>(S_ID, devPtrS));
    data_ptrs.insert(std::pair<uint64_t, void *>(dO_ID, devPtrdO));
    data_ptrs.insert(std::pair<uint64_t, void *>(dS_ID, devPtrdS));
    data_ptrs.insert(
        std::pair<uint64_t, void *>(Q_SEQLEN_ID, devActualSeqlenQ));
    data_ptrs.insert(
        std::pair<uint64_t, void *>(K_SEQLEN_ID, devActualSeqlenKV));

    float zeroVal = 0.0f;
    float dropoutScale = 1.0f / (1.0f - dropout_probability);

    data_ptrs.insert(std::pair<uint64_t, void *>(D_CONST_ID, &dropoutScale));
    data_ptrs.insert(std::pair<uint64_t, void *>(S_CONST_ID, &scaling_factor));
    data_ptrs.insert(std::pair<uint64_t, void *>(MASK_VAL_ID, &zeroVal));

    auto variantPack = cudnn_frontend::VariantPackBuilder()
                           .setWorkspacePointer(workspace_ptr)
                           .setDataPointers(data_ptrs)
                           .build();
    VLOG(10) << "variantPack " << variantPack.describe();
    hipdnnStatus_t status = phi::dynload::cudnnBackendExecute(
        handle_, plan.get_raw_desc(), variantPack.get_raw_desc());
    if (workspace_size > 0) {
      PADDLE_ENFORCE_GPU_SUCCESS(hipFree(workspace_ptr));
    }

    // checkCudnnErr(hipdnnDestroy(handle_));

    cudnn_frontend::throw_if(
        [status]() { return (status != HIPDNN_STATUS_SUCCESS); },
        "Plan execute error",
        status);
  } catch (cudnn_frontend::cudnnException &e) {
    struct hipDeviceProp_t prop;
    PADDLE_ENFORCE_GPU_SUCCESS(hipGetDeviceProperties(&prop, 0));

    // this example is only for GA100 cards and GH100 cards
    if (!((prop.major == 8 && prop.minor == 0) ||
          (prop.major == 9 && prop.minor == 0 && CUDNN_VERSION >= 8800)) &&
        (e.getCudnnStatus() == HIPDNN_STATUS_ARCH_MISMATCH ||
         e.getCudnnStatus() == HIPDNN_STATUS_NOT_SUPPORTED)) {
      std::cout << "Only supported for GA100 (cuDNN >= 8700) and "
                   "GH100 (cuDNN >= 8800) GPUs"
                << std::endl;
    } else {
      std::cout << "[ERROR] Exception " << e.what() << std::endl;
      // CHECK(false);
    }
  }
}

#endif
