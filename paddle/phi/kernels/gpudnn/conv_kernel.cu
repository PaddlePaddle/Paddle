#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/conv_kernel.h"

#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/kernel_registry.h"

#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/conv_miopen_helper.h"
#else
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#endif

#include "paddle/fluid/platform/cudnn_workspace_helper.h"
#include "paddle/fluid/platform/profiler.h"
#include "paddle/phi/common/bfloat16.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/kernels/cpu/conv_util.h"
#include "paddle/phi/kernels/funcs/batch_norm_utils.h"
#include "paddle/phi/kernels/funcs/padding.h"
#include "paddle/phi/kernels/impl/conv_cudnn_impl.h"

namespace phi {

template <typename T, typename Context>
void ConvCudnnKernel(const Context& ctx,
                     const DenseTensor& input,
                     const DenseTensor& filter,
                     const std::vector<int>& strides,
                     const std::vector<int>& paddings_t,
                     const std::string& padding_algorithm,
                     int groups,
                     const std::vector<int>& dilations_t,
                     const std::string& data_format,
                     bool use_addto,
                     int workspace_size_MB,
                     bool exhaustive_search_t,
                     DenseTensor* output) {
  ctx.template Alloc<T>(output);
  std::vector<int> paddings = paddings_t;
  std::vector<int> dilations = dilations_t;

  bool exhaustive_search = FLAGS_cudnn_exhaustive_search || exhaustive_search_t;
  bool deterministic = FLAGS_cudnn_deterministic;
  PADDLE_ENFORCE_EQ(exhaustive_search && deterministic,
                    false,
                    phi::errors::InvalidArgument(
                        "Cann't set exhaustive_search True and "
                        "FLAGS_cudnn_deterministic True at same time."));

  const bool channel_last = (data_format == "NHWC" || data_format == "NDHWC");
  auto dtype = paddle::platform::CudnnDataType<T>::type;

#ifdef PADDLE_WITH_HIP
  // HIP MIOPEN ONLY SUPPORT NCHW format
  auto compute_format = paddle::platform::DataLayout::kNCHW;
#else
  // Tensor Core introduced from Volta GPUs supports more faster conv op
  // with FP16 in NHWC data format.
  const bool compute_in_nhwc = dtype == HIPDNN_DATA_HALF && IsVoltaOrLater(ctx);
  // We will only do data format conversion from NHWC to NCHW.
  // cudnn will convert NCHW to NHWC automatically on Tensor Core.
  auto compute_format = compute_in_nhwc && channel_last
                            ? paddle::platform::DataLayout::kNHWC
                            : paddle::platform::DataLayout::kNCHW;
#endif
  VLOG(3) << "Compute ConvOp with cuDNN:"
          << " data_format=" << data_format << " compute_format="
          << (compute_format == paddle::platform::DataLayout::kNHWC ? "NHWC"
                                                                    : "NCHW");

  // ------------ transformed tensor -----------
  DenseTensor transformed_input_channel(input.type());
  DenseTensor transformed_output(output->type());
  DenseTensor transformed_filter_channel(filter.type());
  T* output_data = nullptr;
  if (channel_last && compute_format == paddle::platform::DataLayout::kNCHW) {
    VLOG(3) << "Transform input tensor from NHWC to NCHW.";
    ResizeToChannelFirst<Context, T>(ctx, &input, &transformed_input_channel);
    TransToChannelFirst<Context, T>(ctx, &input, &transformed_input_channel);

    ResizeToChannelFirst<Context, T>(ctx, output, &transformed_output);

  } else {
    transformed_input_channel.ShareDataWith(input);
    transformed_output.ShareDataWith(*output);
  }
  if (compute_format == paddle::platform::DataLayout::kNHWC) {
    VLOG(3) << "Transform filter tensor from NCHW to NHWC.";
    ResizeToChannelLast<Context, T>(ctx, &filter, &transformed_filter_channel);
    TransToChannelLast<Context, T>(ctx, &filter, &transformed_filter_channel);
  } else {
    transformed_filter_channel.ShareDataWith(filter);
  }
  output_data = transformed_output.data<T>();

  // update padding and dilation
  auto in_dims = transformed_input_channel.dims();
  auto filter_dims = transformed_filter_channel.dims();
  DDim in_data_dims;
  DDim filter_data_dims;

  if (compute_format == paddle::platform::DataLayout::kNCHW) {
    in_data_dims = slice_ddim(in_dims, 2, in_dims.size());
    filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  } else {
    in_data_dims = slice_ddim(in_dims, 1, in_dims.size() - 1);
    filter_data_dims = slice_ddim(filter_dims, 1, filter_dims.size() - 1);
  }

  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings, &dilations, padding_algorithm, in_data_dims, strides, ksize);

  int data_dim = strides.size();  // 2d or 3d
  bool is_sys_pad = funcs::IsSymmetricPadding(paddings, data_dim);

  DenseTensor transformed_input;
  std::vector<int> padding_common(data_dim, 0);
  if (!is_sys_pad) {
    std::vector<int> padding_diff(data_dim);
    std::vector<int> new_input_shape_vec(data_dim + 2);
    new_input_shape_vec[0] = transformed_input_channel.dims()[0];

    if (compute_format == paddle::platform::DataLayout::kNCHW) {
      new_input_shape_vec[1] = transformed_input_channel.dims()[1];
    } else {
      new_input_shape_vec[data_dim + 1] =
          transformed_input_channel.dims()[data_dim + 1];
    }

    std::vector<int> input_pad(transformed_input_channel.dims().size() * 2, 0);
    for (size_t i = 0; i < data_dim; ++i) {
      padding_diff[i] = std::abs(paddings[2 * i] - paddings[2 * i + 1]);
      padding_common[i] = std::min(paddings[2 * i], paddings[2 * i + 1]);
      if (compute_format == paddle::platform::DataLayout::kNCHW) {
        new_input_shape_vec[i + 2] =
            transformed_input_channel.dims()[i + 2] + padding_diff[i];
      } else {
        new_input_shape_vec[i + 1] =
            transformed_input_channel.dims()[i + 1] + padding_diff[i];
      }
      if (compute_format == paddle::platform::DataLayout::kNCHW) {
        input_pad[2 * i + 4] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 4 + 1] = paddings[2 * i + 1] - padding_common[i];
      } else {
        input_pad[2 * i + 2] = paddings[2 * i] - padding_common[i];
        input_pad[2 * i + 2 + 1] = paddings[2 * i + 1] - padding_common[i];
      }
    }
    DDim new_input_shape(make_ddim(new_input_shape_vec));
    transformed_input.Resize(new_input_shape);
    ctx.template Alloc<T>(&transformed_input);

    const int rank = transformed_input_channel.dims().size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(ctx,
                                          input_pad,
                                          transformed_input_channel,
                                          pad_value,
                                          &transformed_input);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(ctx,
                                          input_pad,
                                          transformed_input_channel,
                                          pad_value,
                                          &transformed_input);
      } break;
      default:
        PADDLE_THROW(phi::errors::InvalidArgument(
            "ConvOp only support tensors with 4 or 5 dimensions."));
    }

  } else {
    transformed_input.ShareDataWith(transformed_input_channel);
    if (paddings.size() == data_dim) {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings[i];
      }
    } else {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings[2 * i];
      }
    }
  }

  const T* input_data = transformed_input.data<T>();

  const T* filter_data = transformed_filter_channel.data<T>();

  // ------------------- cudnn descriptors ---------------------
  paddle::operators::ConvArgs args{&transformed_input,
                                   &transformed_filter_channel,
                                   &transformed_output,
                                   strides,
                                   padding_common,
                                   dilations,
                                   dtype,
                                   groups,
                                   compute_format};

  auto handle = ctx.cudnn_handle();
  auto workspace_handle = ctx.cudnn_workspace_handle();
  paddle::platform::DataLayout layout =
      compute_format == paddle::platform::DataLayout::kNHWC
          ? paddle::platform::DataLayout::kNHWC
          : paddle::platform::DataLayout::kNCHW;
  if (transformed_input.dims().size() == 5) {
    layout = compute_format == paddle::platform::DataLayout::kNHWC
                 ? paddle::platform::DataLayout::kNDHWC
                 : paddle::platform::DataLayout::kNCDHW;
  }
  auto layout_format = paddle::platform::GetCudnnTensorFormat(layout);

  args.handle = handle;

#ifdef PADDLE_WITH_HIP
  // MIOPEN need to set groups in cdesc in miopen_desc.h
  args.cdesc.set(dtype,
                 padding_common,
                 strides,
                 dilations,
                 paddle::platform::AllowTF32Cudnn(),
                 groups);
#else
  args.cdesc.set(dtype,
                 padding_common,
                 strides,
                 dilations,
                 paddle::platform::AllowTF32Cudnn());
#endif

#if defined(PADDLE_WITH_CUDA) && CUDNN_VERSION_MIN(7, 0, 1)
  // cudnn 7 can support groups, no need to do it manually
  // FIXME(typhoonzero): find a better way to disable groups
  // rather than setting it to 1.
  PADDLE_ENFORCE_GPU_SUCCESS(
      paddle::platform::dynload::hipdnnSetConvolutionGroupCount(
          args.cdesc.desc(), groups));
  groups = 1;
#endif
#ifdef PADDLE_WITH_HIP
  // MIOPEN do not set groups in wdesc after set groups in cdesc
  groups = 1;
#endif
  args.idesc.set(transformed_input, layout_format);
  args.wdesc.set(transformed_filter_channel, layout_format, groups);
  args.odesc.set(transformed_output, layout_format);
  int i_n, i_c, i_d, i_h, i_w;
  int o_n, o_c, o_d, o_h, o_w;

  if (compute_format == paddle::platform::DataLayout::kNHWC) {
    paddle::operators::GetNCDHW(transformed_input.dims(),
                                paddle::platform::DataLayout::kNHWC,
                                &i_n,
                                &i_c,
                                &i_d,
                                &i_h,
                                &i_w);
    paddle::operators::GetNCDHW(transformed_output.dims(),
                                paddle::platform::DataLayout::kNHWC,
                                &o_n,
                                &o_c,
                                &o_d,
                                &o_h,
                                &o_w);
  } else {
    paddle::operators::GetNCDHW(transformed_input.dims(),
                                paddle::platform::DataLayout::kNCHW,
                                &i_n,
                                &i_c,
                                &i_d,
                                &i_h,
                                &i_w);
    paddle::operators::GetNCDHW(transformed_output.dims(),
                                paddle::platform::DataLayout::kNCHW,
                                &o_n,
                                &o_c,
                                &o_d,
                                &o_h,
                                &o_w);
  }

  int group_offset_in = i_c / groups * i_h * i_w * i_d;
  int group_offset_out = o_c / groups * o_h * o_w * o_d;
  int group_offset_filter = transformed_filter_channel.numel() / groups;
  // ------------------- cudnn conv workspace ---------------------
  size_t workspace_size = 0;  // final workspace to allocate.
// ------------------- cudnn conv algorithm ---------------------
#ifdef PADDLE_WITH_HIP
  paddle::operators::SearchResult<miopenConvFwdAlgorithm_t> fwd_result;
  using search = paddle::operators::SearchAlgorithm<miopenConvFwdAlgorithm_t>;
  workspace_size = search::GetWorkspaceSize(args);
  fwd_result.algo = search::Find<T>(
      args, exhaustive_search, deterministic, workspace_size, ctx);
#else
  paddle::operators::SearchResult<hipdnnConvolutionFwdAlgo_t> fwd_result;
  using search =
      paddle::operators::SearchAlgorithm<hipdnnConvolutionFwdAlgoPerf_t>;
  fwd_result = search::Find<T>(ctx, args, exhaustive_search, deterministic);
  workspace_size = fwd_result.workspace_size;
#endif

#if defined(PADDLE_WITH_CUDA) && CUDNN_VERSION_MIN(7, 0, 1)
  // when groups > 1, SearchAlgorithm find algo is CUDNN_CONVOLUTION_\
    // FWD_ALGO_WINOGRAD_NONFUSED, but this kind of algorithm is unstable
  // in forward computation, so change the algorithm to CUDNN_CONVOLUTION_\
    // FWD_ALGO_IMPLICIT_GEMM manually.
  if (groups > 1) {
    fwd_result.algo = static_cast<hipdnnConvolutionFwdAlgo_t>(0);
  }
#endif

  // ------------------- cudnn conv forward ---------------------
  paddle::operators::ScalingParamType<T> alpha = 1.0f;
  paddle::operators::ScalingParamType<T> beta = 0.0f;

  // NOTE(zhiqiu): inplace addto is not supportted in double grad yet.
  // ScalingParamType<T> beta = ctx.Attr<bool>("use_addto") ? 1.0f : 0.0f;
  // VLOG(4) << "Conv: use_addto = " << ctx.Attr<bool>("use_addto");

#ifdef PADDLE_WITH_HIP
  workspace_handle.RunFunc(
      [&](void* workspace_ptr) {
        PADDLE_ENFORCE_GPU_SUCCESS(
            paddle::platform::dynload::miopenConvolutionForward(
                handle,
                &alpha,
                args.idesc.desc(),
                input_data,
                args.wdesc.desc(),
                filter_data,
                args.cdesc.desc(),
                fwd_result.algo,
                &beta,
                args.odesc.desc(),
                output_data,
                workspace_ptr,
                workspace_size));
      },
      workspace_size);
#else
  for (int i = 0; i < groups; i++) {
    workspace_handle.RunFunc(
        [&](void* workspace_ptr) {
          PADDLE_ENFORCE_GPU_SUCCESS(
              paddle::platform::dynload::hipdnnConvolutionForward(
                  handle,
                  &alpha,
                  args.idesc.desc(),
                  input_data + i * group_offset_in,
                  args.wdesc.desc(),
                  filter_data + i * group_offset_filter,
                  args.cdesc.desc(),
                  fwd_result.algo,
                  workspace_ptr,
                  workspace_size,
                  &beta,
                  args.odesc.desc(),
                  output_data + i * group_offset_out));
        },
        workspace_size);
  }
#endif

  if (channel_last && compute_format == paddle::platform::DataLayout::kNCHW) {
    TransToChannelLast<Context, T>(ctx, &transformed_output, output);
  }
}

template <typename T, typename Context>
void Conv3DCudnnKernel(const Context& dev_ctx,
                       const DenseTensor& input,
                       const DenseTensor& filter,
                       const std::vector<int>& strides,
                       const std::vector<int>& paddings,
                       const std::string& padding_algorithm,
                       int groups,
                       const std::vector<int>& dilations,
                       const std::string& data_format,
                       bool use_addto,
                       int workspace_size_MB,
                       bool exhaustive_search,
                       DenseTensor* out) {
  ConvCudnnKernel<T>(dev_ctx,
                     input,
                     filter,
                     strides,
                     paddings,
                     padding_algorithm,
                     groups,
                     dilations,
                     data_format,
                     use_addto,
                     workspace_size_MB,
                     exhaustive_search,
                     out);
}

template <typename T, typename Context>
void DepthwiseConvCudnnKernel(const Context& dev_ctx,
                              const DenseTensor& input,
                              const DenseTensor& filter,
                              const std::vector<int>& strides,
                              const std::vector<int>& paddings,
                              const std::string& padding_algorithm,
                              int groups,
                              const std::vector<int>& dilations,
                              const std::string& data_format,
                              bool use_addto,
                              int workspace_size_MB,
                              bool exhaustive_search,
                              bool fuse_relu,
                              DenseTensor* out) {
  ConvCudnnKernel<T>(dev_ctx,
                     input,
                     filter,
                     strides,
                     paddings,
                     padding_algorithm,
                     groups,
                     dilations,
                     data_format,
                     use_addto,
                     workspace_size_MB,
                     exhaustive_search,
                     out);
}

}  // namespace phi

#ifdef PADDLE_WITH_HIP
PD_REGISTER_KERNEL(conv2d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnKernel,
                   float,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(conv3d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnKernel,
                   float,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(depthwise_conv2d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::DepthwiseConvCudnnKernel,
                   float,
                   phi::dtype::float16) {}

#else
#if CUDNN_VERSION_MIN(8, 1, 0)
PD_REGISTER_KERNEL(conv2d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}

PD_REGISTER_KERNEL(conv3d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnKernel,
                   float,
                   double,
                   phi::dtype::float16,
                   phi::dtype::bfloat16) {}
#else
PD_REGISTER_KERNEL(conv2d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::ConvCudnnKernel,
                   float,
                   double,
                   phi::dtype::float16) {}

PD_REGISTER_KERNEL(conv3d,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3DCudnnKernel,
                   float,
                   double,
                   phi::dtype::float16) {}
#endif

#endif

// todo register bfloat16
