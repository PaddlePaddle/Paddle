#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/conv_transpose_kernel.h"

#include <algorithm>

#include "paddle/phi/backends/dynload/hipDNN.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/ddim.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/cpu/conv_util.h"
#include "paddle/phi/kernels/funcs/padding.h"
#include "paddle/phi/kernels/funcs/slice.h"
#include "paddle/phi/kernels/transpose_kernel.h"

#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/operators/conv_miopen_helper.h"
#include "paddle/fluid/platform/device/gpu/rocm/miopen_helper.h"
#else
#include "paddle/fluid/operators/conv_cudnn_helper.h"
#include "paddle/fluid/platform/device/gpu/cuda/cudnn_helper.h"
#endif

namespace phi {

using GPUDNNDataLayout = paddle::platform::DataLayout;

template <typename T, typename Context>
void ConvTransposeRawGPUDNNKernel(const Context& ctx,
                                  const DenseTensor& x,
                                  const DenseTensor& filter,
                                  const std::vector<int>& strides,
                                  const std::vector<int>& paddings,
                                  const std::string& padding_algorithm,
                                  int groups,
                                  const std::vector<int>& dilations,
                                  const std::string& data_format,
                                  DenseTensor* out) {
  std::vector<int> paddings_ = paddings;
  std::vector<int> dilations_ =
      dilations;  // cudnn v5 does not support dilations
  const T* filter_data = filter.data<T>();
  const GPUDNNDataLayout data_layout =
      (data_format != "NHWC" ? GPUDNNDataLayout::kNCHW
                             : GPUDNNDataLayout::kNHWC);
  std::vector<int> x_vec = vectorize<int>(x.dims());
  std::vector<int> out_vec = vectorize<int>(out->dims());
  // if channel_last, transpose to channel_first
  DenseTensor x_transpose;
  if (data_layout == GPUDNNDataLayout::kNHWC) {
    if (strides.size() == 2U) {
      std::vector<int> axis = {0, 3, 1, 2};
      for (size_t i = 0; i < axis.size(); ++i) {
        x_vec[i] = x.dims()[axis[i]];
        out_vec[i] = out->dims()[axis[i]];
      }
      x_transpose = Transpose<T, Context>(ctx, x, axis);
    } else if (strides.size() == 3U) {
      std::vector<int> axis = {0, 4, 1, 2, 3};
      for (size_t i = 0; i < axis.size(); ++i) {
        x_vec[i] = x.dims()[axis[i]];
        out_vec[i] = out->dims()[axis[i]];
      }
      x_transpose = Transpose<T, Context>(ctx, x, axis);
    }
  } else {
    x_transpose = x;
  }

  // update padding and dilation
  auto x_dims = x_transpose.dims();
  auto filter_dims = filter.dims();
  DDim x_data_dims;
  x_data_dims = slice_ddim(x_dims, 2, x_dims.size());
  DDim filter_data_dims = slice_ddim(filter_dims, 2, filter_dims.size());
  std::vector<int> ksize = vectorize<int>(filter_data_dims);
  UpdatePaddingAndDilation(
      &paddings_, &dilations_, padding_algorithm, x_data_dims, strides, ksize);

  int data_dim = strides.size();  // 2d or 3d
  bool is_sys_pad = funcs::IsSymmetricPadding(paddings_, data_dim);

  std::vector<int> x_pad(x_dims.size() * 2, 0);
  DenseTensor transformed_x;
  std::vector<int> padding_common(data_dim, 0);
  if (!is_sys_pad) {
    std::vector<int> padding_diff(data_dim);
    std::vector<int> new_x_shape_vec(data_dim + 2);
    new_x_shape_vec[0] = x_dims[0];
    new_x_shape_vec[1] = x_dims[1];

    for (size_t i = 0; i < data_dim; ++i) {
      padding_diff[i] = std::abs(paddings_[2 * i] - paddings_[2 * i + 1]);
      padding_common[i] = std::min(paddings_[2 * i], paddings_[2 * i + 1]);
      new_x_shape_vec[i + 2] = x_dims[i + 2] + padding_diff[i];
      x_pad[2 * i + 4] = paddings_[2 * i] - padding_common[i];
      x_pad[2 * i + 4 + 1] = paddings_[2 * i + 1] - padding_common[i];
    }
    DDim new_x_shape(make_ddim(new_x_shape_vec));
    transformed_x.Resize(new_x_shape);
    ctx.template Alloc<T>(&transformed_x);

    const int rank = x_dims.size();
    T pad_value(0.0);
    switch (rank) {
      case 4: {
        funcs::PadFunction<Context, T, 4>(
            ctx, x_pad, x_transpose, pad_value, &transformed_x);
      } break;
      case 5: {
        funcs::PadFunction<Context, T, 5>(
            ctx, x_pad, x_transpose, pad_value, &transformed_x);
      } break;
      default:
        PADDLE_THROW(errors::InvalidArgument(
            "Op(ConvTranspose) only supports 4-D or 5-D x DenseTensor."));
    }
  } else {
    transformed_x = x_transpose;
    if (paddings_.size() == data_dim) {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings_[i];
      }
    } else {
      for (size_t i = 0; i < data_dim; ++i) {
        padding_common[i] = paddings_[2 * i];
      }
    }
  }

  std::vector<int64_t> starts(data_dim, 0);
  std::vector<int64_t> ends(data_dim, 0);
  std::vector<int64_t> axes(data_dim, 0);
  for (size_t i = 0; i < data_dim; ++i) {
    starts[i] = x_pad[2 * i + 4] * (strides[i] + 1);
    ends[i] = starts[i] + out_vec[i + 2];
    axes[i] = i + 2;
  }

  const T* x_data = transformed_x.data<T>();
  x_vec = vectorize<int>(transformed_x.dims());

  std::vector<int> transformed_out_vec = out_vec;
  for (size_t i = 0; i < data_dim; ++i) {
    transformed_out_vec[i + 2] =
        out_vec[i + 2] + (x_pad[2 * i + 4] + x_pad[2 * i + 5]) * strides[i] -
        2 * padding_common[i] + paddings_[2 * i] + paddings_[2 * i + 1];
  }

  DenseTensor transformed_out;
  if (!is_sys_pad) {
    transformed_out.Resize(make_ddim(transformed_out_vec));
    ctx.template Alloc<T>(&transformed_out);
  } else {
    ctx.template Alloc<T>(out);
    transformed_out.ShareDataWith(*out);
    transformed_out.Resize(make_ddim(transformed_out_vec));
  }
  T* transformed_out_data = transformed_out.data<T>();

  GPUDNNDataLayout layout;

  int iwo_groups = groups;
  int c_groups = 1;
#if defined(PADDLE_WITH_HIP) || CUDNN_VERSION_MIN(7, 0, 1)
  iwo_groups = 1;
  c_groups = groups;
  groups = 1;
#endif

  if (strides.size() == 2U) {
    layout = GPUDNNDataLayout::kNCHW;
  } else {
    layout = GPUDNNDataLayout::kNCDHW;
  }

  size_t workspace_size = 0;
#ifdef PADDLE_WITH_HIP
  miopenConvBwdDataAlgorithm_t algo{};
#else
  hipdnnConvolutionBwdDataAlgo_t algo{};
#endif
  // ------------------- cudnn conv algorithm ---------------------
  auto handle = ctx.cudnn_handle();
  auto layout_tensor = paddle::platform::GetCudnnTensorFormat(layout);
  bool deterministic = FLAGS_cudnn_deterministic;

  auto dtype = paddle::platform::CudnnDataType<T>::type;
  // ------------------- cudnn descriptors ---------------------
  paddle::operators::ConvArgs args{&transformed_out,
                                   &filter,
                                   &transformed_x,
                                   strides,
                                   padding_common,
                                   dilations_,
                                   dtype,
                                   groups,
                                   data_layout};
  args.handle = handle;
  args.idesc.set(transformed_out, iwo_groups);
  args.wdesc.set(filter, layout_tensor, iwo_groups);
  args.odesc.set(transformed_x, iwo_groups);
  args.cdesc.set(dtype,
                 padding_common,
                 strides,
                 dilations_,
                 paddle::platform::AllowTF32Cudnn(),
                 c_groups);

#ifdef PADDLE_WITH_HIP
  paddle::operators::SearchResult<miopenConvBwdDataAlgorithm_t> bwd_result;
  using search =
      paddle::operators::SearchAlgorithm<miopenConvBwdDataAlgorithm_t>;
  workspace_size = std::max(workspace_size, search::GetWorkspaceSize(args));
  bwd_result.algo =
      search::Find<T>(args, false, deterministic, workspace_size, ctx);
#else
  paddle::operators::SearchResult<hipdnnConvolutionBwdDataAlgo_t> bwd_result;
  using search =
      paddle::operators::SearchAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>;
  bwd_result = search::Find<T>(ctx, args, false, deterministic, false);
  workspace_size =
      std::max(workspace_size, search::GetWorkspaceSize(args, bwd_result.algo));
#endif

  // ------------------- cudnn conv transpose forward ---------------------
  int x_offset = transformed_x.numel() / transformed_x.dims()[0] / groups;
  int out_offset = transformed_out.numel() / transformed_out.dims()[0] / groups;
  int filter_offset = filter.numel() / groups;
  paddle::operators::ScalingParamType<T> alpha = 1.0f;
  paddle::operators::ScalingParamType<T> beta = 0.0f;
  auto workspace_handle = ctx.cudnn_workspace_handle();
  for (int g = 0; g < groups; g++) {
#ifdef PADDLE_WITH_HIP
    auto cudnn_func = [&](void* cudnn_workspace) {
      PADDLE_ENFORCE_GPU_SUCCESS(dynload::miopenConvolutionBackwardData(
          handle,
          &alpha,
          args.odesc.desc(),
          x_data + x_offset * g,
          args.wdesc.desc(),
          filter_data + filter_offset * g,
          args.cdesc.desc(),
          bwd_result.algo,
          &beta,
          args.idesc.desc(),
          transformed_out_data + out_offset * g,
          cudnn_workspace,
          workspace_size));
    };
#else   // PADDLE_WITH_HIP
    auto cudnn_func = [&](void* cudnn_workspace) {
      PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipdnnConvolutionBackwardData(
          handle,
          &alpha,
          args.wdesc.desc(),
          filter_data + filter_offset * g,
          args.odesc.desc(),
          x_data + x_offset * g,
          args.cdesc.desc(),
          bwd_result.algo,
          cudnn_workspace,
          workspace_size,
          &beta,
          args.idesc.desc(),
          transformed_out_data + out_offset * g));
    };
#endif  // PADDLE_WITH_HIP
    workspace_handle.RunFunc(cudnn_func, workspace_size);
  }
  if (!is_sys_pad && strides.size() == 2U) {
    funcs::Slice<Context, T, 4>(ctx, &transformed_out, out, starts, ends, axes);
  } else if (!is_sys_pad && strides.size() == 3U) {
    funcs::Slice<Context, T, 5>(ctx, &transformed_out, out, starts, ends, axes);
  }

  if (data_layout == GPUDNNDataLayout::kNHWC) {
    DenseTensor out_transpose;
    DenseTensor out_nchw;
    out_nchw.ShareDataWith(*out);
    out_nchw.Resize(make_ddim(out_vec));

    if (strides.size() == 2U) {
      out_transpose = Transpose<T, Context>(ctx, out_nchw, {0, 2, 3, 1});
    } else if (strides.size() == 3U) {
      out_transpose = Transpose<T, Context>(ctx, out_nchw, {0, 2, 3, 4, 1});
    }
    *out = out_transpose;
  }
}

template <typename T, typename Context>
void Conv2dTransposeGPUDNNKernel(const Context& ctx,
                                 const DenseTensor& x,
                                 const DenseTensor& filter,
                                 const std::vector<int>& strides,
                                 const std::vector<int>& paddings,
                                 const std::vector<int>& output_padding,
                                 const IntArray& output_size,
                                 const std::string& padding_algorithm,
                                 int groups,
                                 const std::vector<int>& dilations,
                                 const std::string& data_format,
                                 DenseTensor* out) {
  ConvTransposeRawGPUDNNKernel<T, Context>(ctx,
                                           x,
                                           filter,
                                           strides,
                                           paddings,
                                           padding_algorithm,
                                           groups,
                                           dilations,
                                           data_format,
                                           out);
}

template <typename T, typename Context>
void Conv3dTransposeGPUDNNKernel(const Context& ctx,
                                 const DenseTensor& x,
                                 const DenseTensor& filter,
                                 const std::vector<int>& strides,
                                 const std::vector<int>& paddings,
                                 const std::vector<int>& output_padding,
                                 const std::vector<int>& output_size,
                                 const std::string& padding_algorithm,
                                 int groups,
                                 const std::vector<int>& dilations,
                                 const std::string& data_format,
                                 DenseTensor* out) {
  ConvTransposeRawGPUDNNKernel<T, Context>(ctx,
                                           x,
                                           filter,
                                           strides,
                                           paddings,
                                           padding_algorithm,
                                           groups,
                                           dilations,
                                           data_format,
                                           out);
}

}  // namespace phi

using float16 = phi::dtype::float16;

#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
PD_REGISTER_KERNEL(conv2d_transpose,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv2dTransposeGPUDNNKernel,
                   float,
                   float16) {}
PD_REGISTER_KERNEL(conv3d_transpose,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3dTransposeGPUDNNKernel,
                   float,
                   float16) {}
#else
PD_REGISTER_KERNEL(conv2d_transpose,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv2dTransposeGPUDNNKernel,
                   float,
                   double,
                   float16) {}
PD_REGISTER_KERNEL(conv3d_transpose,
                   GPUDNN,
                   ALL_LAYOUT,
                   phi::Conv3dTransposeGPUDNNKernel,
                   float,
                   double,
                   float16) {}
#endif
