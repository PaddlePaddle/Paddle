#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/selected_rows/adagrad_kernel.h"

#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/kernels/selected_rows/impl/adagrad_kernel_impl.h"

namespace phi {
namespace sr {

template <typename T, int block_size>
__global__ void SparseAdagradFunctorKernel(const T* grad,
                                           const int64_t* rows,
                                           const T* learning_rate,
                                           T* param,
                                           T* moment,
                                           int64_t row_numel,
                                           T epsilon) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  grad += ty * row_numel;
  param += rows[ty] * row_numel;
  moment += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(param + index,
                                    -1.0 * learning_rate[0] * grad[index] /
                                        (sqrt(moment[index]) + epsilon));
  }
}

template <typename T>
struct SparseAdagradFunctor<phi::GPUContext, T> {
  void operator()(const phi::GPUContext& context,
                  const phi::SelectedRows& grad,
                  const DenseTensor& learning_rate,
                  T epsilon,
                  DenseTensor* moment,
                  DenseTensor* param) {
    // 1. g_m.rows = set(g.rows)
    auto grad_width = grad.value().dims()[1];
    paddle::operators::math::scatter::MergeAdd<phi::GPUContext, T> merge_func;
    auto grad_merge = merge_func(context, grad);
    auto* grad_merge_data = grad_merge.mutable_value()->template data<T>();
    paddle::framework::Vector<int64_t> merge_rows(grad_merge.rows());
    // 2. m += g_m * g_m
    auto grad_square =
        SquareSelectedRows<phi::GPUContext, T>(context, grad_merge);

    paddle::operators::math::SelectedRowsAddToTensor<phi::GPUContext, T>
        functor;
    functor(context, grad_square, moment);

    // 3. update parameter
    auto* lr = learning_rate.data<T>();
    auto* param_data = param->data<T>();
    auto* moment_data = moment->data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid2(1, merge_rows.size());
    paddle::framework::MixVector<int64_t> mixv_merge_rows(&merge_rows);
    SparseAdagradFunctorKernel<
        T,
        256><<<grid2,
               threads,
               0,
               reinterpret_cast<const phi::GPUContext&>(context).stream()>>>(
        grad_merge_data,
        mixv_merge_rows.CUDAMutableData(context.GetPlace()),
        lr,
        param_data,
        moment_data,
        grad_width,
        epsilon);
    mixv_merge_rows.CopyToCPU();
  }
};

template struct SparseAdagradFunctor<phi::GPUContext, float>;
template struct SparseAdagradFunctor<phi::GPUContext, double>;

}  // namespace sr
}  // namespace phi

PD_REGISTER_KERNEL(
    adagrad_sr, GPU, ALL_LAYOUT, phi::sr::AdagradSparseKernel, float, double) {}
