#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/selected_rows/adam_kernel.h"

#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/operators/math/selected_rows_functor.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/core/tensor_utils.h"
#include "paddle/phi/kernels/funcs/adam_functors.h"
#include "paddle/phi/kernels/funcs/for_range.h"

namespace phi {
namespace sr {

template <typename T>
__global__ void UpdateBetaPow(T beta1,
                              T beta2,
                              const T* beta1_pow_,
                              const T* beta2_pow_,
                              T* beta1_pow_out,
                              T* beta2_pow_out) {
  *beta1_pow_out = beta1 * beta1_pow_[0];
  *beta2_pow_out = beta2 * beta2_pow_[0];
}

template <typename T, typename MT>
__global__ void SparseAdamCUDAKernelREG(MT beta1,
                                        MT beta2,
                                        MT epsilon,
                                        const MT beta1_pow,
                                        const MT beta2_pow,
                                        const MT* mom1_,
                                        MT* mom1_out_,
                                        const MT* mom2_,
                                        MT* mom2_out_,
                                        const MT* lr_,
                                        const T* grad_,
                                        const T* param_,
                                        T* param_out_,
                                        const MT* master_param,
                                        MT* master_param_out,
                                        const int64_t* rows_,
                                        int64_t row_numel,
                                        int64_t row_count,
                                        bool lazy_mode,
                                        int ndim) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  MT lr = *lr_;

  for (; id < ndim; id += blockDim.x * gridDim.x) {
    auto row_idx =
        phi::funcs::BinarySearch<int64_t>(rows_, row_count, id / row_numel);
    if (lazy_mode && row_idx < 0) {
      return;
    } else {
      MT mom1 = mom1_[id];
      MT mom2 = mom2_[id];
      MT p = master_param ? master_param[id] : static_cast<MT>(param_[id]);
      MT g = row_idx >= 0
                 ? static_cast<MT>(grad_[row_idx * row_numel + id % row_numel])
                 : static_cast<MT>(0);
      mom1 = beta1 * mom1 + (static_cast<MT>(1.0) - beta1) * g;
      mom2 = beta2 * mom2 + (static_cast<MT>(1.0) - beta2) * g * g;

      MT denom =
          (sqrt(mom2) / sqrt(static_cast<MT>(1.0) - beta2_pow)) + epsilon;
      p += (mom1 / denom) * (-(lr / (static_cast<MT>(1.0) - beta1_pow)));

      // Write back to global memory
      mom1_out_[id] = mom1;
      mom2_out_[id] = mom2;
      param_out_[id] = static_cast<T>(p);
      if (master_param_out) {
        master_param_out[id] = p;
      }
    }
  }
}

template <typename T, typename Context>
void AdamDenseParamSparseGradKernel(
    const Context& dev_ctx,
    const DenseTensor& param,
    const SelectedRows& grad,
    const DenseTensor& learning_rate,
    const DenseTensor& moment1,
    const DenseTensor& moment2,
    const DenseTensor& beta1_pow,
    const DenseTensor& beta2_pow,
    const paddle::optional<DenseTensor>& master_param,
    const paddle::optional<DenseTensor>& skip_update,
    const Scalar& beta1,
    const Scalar& beta2,
    const Scalar& epsilon,
    bool lazy_mode,
    int64_t min_row_size_to_use_multithread,
    bool multi_precision,
    bool use_global_beta_pow,
    DenseTensor* param_out,
    DenseTensor* moment1_out,
    DenseTensor* moment2_out,
    DenseTensor* beta1_pow_out,
    DenseTensor* beta2_pow_out,
    DenseTensor* master_param_outs) {
  using MPDType = typename phi::dtype::MPTypeTrait<T>::Type;

  VLOG(4) << "use_global_beta_pow:" << use_global_beta_pow;

  bool skip_update_ = false;
  if (skip_update.is_initialized()) {
    PADDLE_ENFORCE_EQ(
        skip_update->numel(),
        1,
        errors::InvalidArgument("Input(SkipUpdate) size must be 1, but get %d",
                                skip_update->numel()));
    std::vector<bool> skip_update_vec;
    paddle::framework::TensorToVector(*skip_update, dev_ctx, &skip_update_vec);
    skip_update_ = skip_update_vec[0];
  }
  // skip_update=true, just copy input to output, and TensorCopy will call
  // mutable_data
  if (skip_update_) {
    VLOG(4) << "Adam skip update";
    phi::Copy(dev_ctx, param, dev_ctx.GetPlace(), false, param_out);
    phi::Copy(dev_ctx, moment1, dev_ctx.GetPlace(), false, moment1_out);
    phi::Copy(dev_ctx, moment2, dev_ctx.GetPlace(), false, moment2_out);
    phi::Copy(dev_ctx, beta1_pow, beta1_pow.place(), false, beta1_pow_out);
    phi::Copy(dev_ctx, beta2_pow, beta2_pow.place(), false, beta2_pow_out);
    return;
  }

  MPDType beta1_ = beta1.to<MPDType>();
  MPDType beta2_ = beta2.to<MPDType>();
  MPDType epsilon_ = epsilon.to<MPDType>();
  VLOG(3) << "beta1_pow.numel() : " << beta1_pow.numel()
          << "beta2_pow.numel() : " << beta2_pow.numel();
  VLOG(3) << "param.numel(): " << param.numel();
  PADDLE_ENFORCE_EQ(
      beta1_pow_out->numel(),
      1,
      errors::InvalidArgument("beta1 pow output size should be 1, but received "
                              "value is:%d.",
                              beta1_pow_out->numel()));

  PADDLE_ENFORCE_EQ(
      beta2_pow_out->numel(),
      1,
      errors::InvalidArgument("beta2 pow output size should be 1, but received "
                              "value is:%d.",
                              beta2_pow_out->numel()));

  const MPDType* master_in_data =
      multi_precision ? master_param->data<MPDType>() : nullptr;
  MPDType* master_out_data =
      multi_precision ? dev_ctx.template Alloc<MPDType>(master_param_outs)
                      : nullptr;

  if (grad.rows().size() == 0) {
    VLOG(3) << "grad row size is 0!!";
    return;
  }

  std::vector<int64_t> cpu_rows(grad.rows().begin(), grad.rows().end());
  bool is_strict_sorted = true;
  for (size_t i = 1; i < cpu_rows.size(); ++i) {
    if (cpu_rows[i - 1] >= cpu_rows[i]) {
      is_strict_sorted = false;
      break;
    }
  }

  phi::SelectedRows tmp_grad_merge;
  const phi::SelectedRows* grad_merge_ptr;
  if (is_strict_sorted) {
    grad_merge_ptr = &grad;
  } else {
    // merge duplicated rows if any.
    // The rows of grad_merge have been sorted inside MergeAdd functor
    paddle::operators::math::scatter::MergeAdd<Context, T> merge_func;
    merge_func(dev_ctx, grad, &tmp_grad_merge, true);
    grad_merge_ptr = &tmp_grad_merge;
  }
  auto& grad_merge = *grad_merge_ptr;
  auto& grad_tensor = grad_merge.value();
  const T* grad_data = grad_tensor.template data<T>();
  auto* grad_merge_rows = &grad_merge.rows();
  paddle::framework::MixVector<int64_t> mixv_grad_merge_rows(grad_merge_rows);
  const int64_t* rows = mixv_grad_merge_rows.Data(dev_ctx.GetPlace());
  auto row_numel = grad_tensor.numel() / grad_merge.rows().size();

  if (beta1_pow.place() == CPUPlace() && beta2_pow.place() == CPUPlace()) {
    int threads = 512;
    int ndim = param.numel();
    int blocks = (ndim + threads - 1) / threads;

    SparseAdamCUDAKernelREG<T, MPDType>
        <<<blocks, threads, 0, dev_ctx.stream()>>>(
            beta1_,
            beta2_,
            epsilon_,
            *beta1_pow.data<MPDType>(),
            *beta2_pow.data<MPDType>(),
            moment1.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment1_out),
            moment2.data<MPDType>(),
            dev_ctx.template Alloc<MPDType>(moment2_out),
            learning_rate.data<MPDType>(),
            grad_data,
            param.data<T>(),
            dev_ctx.template Alloc<T>(param_out),
            master_in_data,
            master_out_data,
            rows,
            row_numel,
            grad_merge.rows().size(),
            lazy_mode,
            ndim);
    if (!use_global_beta_pow) {
      // Update with cpu
      dev_ctx.template HostAlloc<MPDType>(beta1_pow_out)[0] =
          beta1_ * beta1_pow.data<MPDType>()[0];
      dev_ctx.template HostAlloc<MPDType>(beta2_pow_out)[0] =
          beta2_ * beta2_pow.data<MPDType>()[0];
    }
  } else {
    funcs::SparseAdamFunctor<T, funcs::GPUAdam, MPDType> functor(
        beta1_,
        beta2_,
        epsilon_,
        beta1_pow.data<MPDType>(),
        beta2_pow.data<MPDType>(),
        moment1.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(moment1_out),
        moment2.data<MPDType>(),
        dev_ctx.template Alloc<MPDType>(moment2_out),
        learning_rate.data<MPDType>(),
        grad_data,
        param.data<T>(),
        dev_ctx.template Alloc<T>(param_out),
        master_in_data,
        master_out_data,
        rows,
        row_numel,
        grad_merge.rows().size(),
        lazy_mode);

    // FIXME(minqiyang): remove BinarySearch in GPU later
    funcs::ForRange<Context> for_range(dev_ctx, param.numel());
    for_range(functor);
    if (!use_global_beta_pow) {
      // update beta1 and beta2
      UpdateBetaPow<MPDType><<<1, 32, 0, dev_ctx.stream()>>>(
          beta1_,
          beta2_,
          beta1_pow.data<MPDType>(),
          beta2_pow.data<MPDType>(),
          dev_ctx.template Alloc<MPDType>(beta1_pow_out),
          dev_ctx.template Alloc<MPDType>(beta2_pow_out));
    }
  }
}

}  // namespace sr
}  // namespace phi

PD_REGISTER_KERNEL(adam_dense_param_sparse_grad,
                   GPU,
                   ALL_LAYOUT,
                   phi::sr::AdamDenseParamSparseGradKernel,
                   float,
                   double,
                   phi::dtype::float16) {
  // Skip beta1_pow, beta2_pow, skip_update data transform
  kernel->InputAt(5).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(6).SetBackend(phi::Backend::ALL_BACKEND);
  kernel->InputAt(8).SetBackend(phi::Backend::ALL_BACKEND);
}
