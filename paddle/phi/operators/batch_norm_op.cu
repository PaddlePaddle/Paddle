/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <cfloat>
#include <string>
#include <vector>
#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif
#include "paddle/common/flags.h"
#include "paddle/fluid/framework/data_layout.h"
#include "paddle/fluid/platform/float16.h"
#include "paddle/phi/backends/gpu/gpu_dnn.h"
#include "paddle/phi/kernels/funcs/math_function.h"
#include "paddle/phi/operators/batch_norm_op.h"

COMMON_DECLARE_bool(cudnn_batchnorm_spatial_persistent);

namespace paddle {
namespace operators {

using DataLayout = phi::DataLayout;
template <typename T>
using CudnnDataType = phi::backends::gpu::CudnnDataType<T>;
template <typename T>
using BatchNormParamType = typename CudnnDataType<T>::BatchNormParamType;

}  // namespace operators
}  // namespace paddle
