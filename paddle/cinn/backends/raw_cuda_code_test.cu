#include "hip/hip_runtime.h"
// Copyright (c) 2021 CINN Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <glog/logging.h>
#include <gtest/gtest.h>

#include "cinn/backends/cuda_util.h"
#include "cinn/utils/timer.h"

__global__ void elementwise_add_kernel(const float* __restrict__ A,
                                       const float* __restrict__ B,
                                       float* __restrict__ C) {
  if ((blockIdx.x < 1024)) {
    {
      if ((threadIdx.x < 1024)) {
        {
          C[((1024 * blockIdx.x) + threadIdx.x)] =
              (A[((1024 * blockIdx.x) + threadIdx.x)] +
               B[((1024 * blockIdx.x) + threadIdx.x)]);
        }
      }
    }
  }
}

TEST(raw_cuda, basic) {
  const int M = 1024;
  const int N = 1024;
  // allocate CUDA buffer
  float *Ag, *Bg, *Cg;
  const int num_bytes = M * N * sizeof(float);
  hipMalloc(&Ag, num_bytes);
  hipMalloc(&Bg, num_bytes);
  hipMalloc(&Cg, num_bytes);

  cinn::utils::Timer timer;
  timer.Start();
  for (int i = 0; i < 1000; i++) {
    elementwise_add_kernel<<<1024, 1024>>>(Ag, Bg, Cg);
  }
  CUDA_CALL(hipDeviceSynchronize());
  float latency = timer.Stop();
  LOG(INFO) << "latency: " << latency / 1000;
}
