#include "hip/hip_runtime.h"
// Copyright (c) 2021 CINN Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <glog/logging.h>
#include <gtest/gtest.h>

#include <random>
#include <vector>
#include "paddle/cinn/common/bfloat16.h"
#include "paddle/cinn/common/float16.h"
#include "paddle/common/enforce.h"

namespace cinn {
namespace common {

#define CUDA_CALL(func)                                    \
  {                                                        \
    auto status = func;                                    \
    if (status != hipSuccess) {                           \
      std::stringstream ss;                                \
      ss << "CUDA Error : " << hipGetErrorString(status); \
      PADDLE_THROW(phi::errors::Fatal(ss.str()));          \
    }                                                      \
  }

class CudaMem {
 public:
  CudaMem() = default;

  void* mutable_data(size_t bytes) {
    CHECK_GT(bytes, 0) << "Cannot allocate empty memory!";
    if (ptr) {
      CHECK_EQ(bytes, bytes_) << "Try allocate memory twice!";
      return ptr;
    }
    CUDA_CALL(hipMalloc(&ptr, bytes));
    bytes_ = bytes;
    return ptr;
  }

  template <typename T>
  T* mutable_data(size_t num) {
    return reinterpret_cast<T*>(mutable_data(num * sizeof(T)));
  }

  void* data() const {
    CHECK(ptr) << "Try get nullptr!";
    return ptr;
  }

  template <typename T>
  T* data() const {
    return reinterpret_cast<T*>(data());
  }

  void MemcpyFromHost(const void* src,
                      size_t bytes,
                      hipStream_t stream = nullptr) {
    CHECK_LE(bytes, bytes_) << "Too many data need copy";
    CUDA_CALL(hipMemcpyAsync(ptr, src, bytes, hipMemcpyHostToDevice, stream));
  }

  void MemcpyToHost(void* dst, size_t bytes, hipStream_t stream = nullptr) {
    CHECK_LE(bytes, bytes_) << "Too many data need copy";
    CUDA_CALL(hipMemcpyAsync(dst, ptr, bytes, hipMemcpyDeviceToHost, stream));
  }

  ~CudaMem() {
    if (ptr) {
      hipFree(ptr);
    }
    bytes_ = 0;
  }

 private:
  void* ptr{nullptr};
  size_t bytes_{0};
};

__global__ void cast_fp32_to_fp16_cuda_kernel(const float* input,
                                              const int num,
                                              float16* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    out[idx] = float16(input[idx]);
  }
}

__global__ void cast_fp16_to_fp32_cuda_kernel(const float16* input,
                                              const int num,
                                              float* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    out[idx] = static_cast<float>(input[idx]);
  }
}

__global__ void test_fp16_cuda_kernel(const float16* x,
                                      const float16* y,
                                      const int num,
                                      float16* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    float16 x_i = x[idx], y_i = y[idx];
    x_i += float16(1);

    out[idx] = (x_i + y_i) * (x_i - y_i);
  }
}

__global__ void cast_fp32_to_bf16_cuda_kernel(const float* input,
                                              const int num,
                                              bfloat16* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    out[idx] = bfloat16(input[idx]);
  }
}

__global__ void cast_bf16_to_fp32_cuda_kernel(const bfloat16* input,
                                              const int num,
                                              float* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    out[idx] = static_cast<float>(input[idx]);
  }
}

__global__ void test_bf16_cuda_kernel(const bfloat16* x,
                                      const bfloat16* y,
                                      const int num,
                                      bfloat16* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    bfloat16 x_i = x[idx], y_i = y[idx];
    x_i += bfloat16(1);

    out[idx] = (x_i + y_i) * (x_i - y_i);
  }
}

__global__ void test_fp32_cuda_kernel(const float* x,
                                      const float* y,
                                      const int num,
                                      float* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    float x_i = x[idx], y_i = y[idx];
    x_i += 1.0f;

    out[idx] = (x_i + y_i) * (x_i - y_i);
  }
}

TEST(FP16_BF16, basic_cuda) {
#ifdef CUDA_VERSION
  LOG(INFO) << "CUDA version: " << CUDA_VERSION;
#endif

  int num = 2048;

  hipStream_t stream;
  CUDA_CALL(hipStreamCreate(&stream));

  dim3 block = 1024;
  dim3 grid = (num + block.x - 1) / block.x;

  std::vector<float> x_fp32_host(num), y_fp32_host(num);
  {  // step1 : generate input data
    std::random_device r;
    std::default_random_engine eng(r());
    std::uniform_real_distribution<float> dis(1e-5f, 1.0f);

    for (int i = 0; i < num; ++i) {
      x_fp32_host[i] = dis(eng);
      y_fp32_host[i] = dis(eng);
    }
  }

  CudaMem x_fp32_device, y_fp32_device, out_fp32_device;
  {  // step2 : compute fp32 result
    auto x_fp32_ptr = x_fp32_device.mutable_data<float>(num);
    auto y_fp32_ptr = y_fp32_device.mutable_data<float>(num);
    auto out_fp32_ptr = out_fp32_device.mutable_data<float>(num);

    x_fp32_device.MemcpyFromHost(
        x_fp32_host.data(), num * sizeof(float), stream);
    y_fp32_device.MemcpyFromHost(
        y_fp32_host.data(), num * sizeof(float), stream);

    test_fp32_cuda_kernel<<<grid, block, 0, stream>>>(
        x_fp32_ptr, y_fp32_ptr, num, out_fp32_ptr);
  }

  CudaMem x_fp16_device, y_fp16_device, out_fp16_device;
  CudaMem x_bf16_device, y_bf16_device, out_bf16_device;
  {  // step3 : compute fp16/bf16 result
    // step3.1 : compute fp16 result
    auto x_fp16_ptr = x_fp16_device.mutable_data<float16>(num);
    auto y_fp16_ptr = y_fp16_device.mutable_data<float16>(num);
    auto out_fp16_ptr = out_fp16_device.mutable_data<float16>(num);

    cast_fp32_to_fp16_cuda_kernel<<<grid, block, 0, stream>>>(
        x_fp32_device.data<float>(), num, x_fp16_ptr);
    cast_fp32_to_fp16_cuda_kernel<<<grid, block, 0, stream>>>(
        y_fp32_device.data<float>(), num, y_fp16_ptr);

    test_fp16_cuda_kernel<<<grid, block, 0, stream>>>(
        x_fp16_ptr, y_fp16_ptr, num, out_fp16_ptr);

    // step3.2 : compute bf16 result
    auto x_bf16_ptr = x_bf16_device.mutable_data<bfloat16>(num);
    auto y_bf16_ptr = y_bf16_device.mutable_data<bfloat16>(num);
    auto out_bf16_ptr = out_bf16_device.mutable_data<bfloat16>(num);

    cast_fp32_to_bf16_cuda_kernel<<<grid, block, 0, stream>>>(
        x_fp32_device.data<float>(), num, x_bf16_ptr);
    cast_fp32_to_bf16_cuda_kernel<<<grid, block, 0, stream>>>(
        y_fp32_device.data<float>(), num, y_bf16_ptr);

    test_bf16_cuda_kernel<<<grid, block, 0, stream>>>(
        x_bf16_ptr, y_bf16_ptr, num, out_bf16_ptr);
  }

  CudaMem fp32res_fp16_device;
  CudaMem fp32res_bf16_device;
  {  // step4 : cast fp16/bf16 result to fp32 result
    // step4.1 : cast fp16 result to fp32 result
    auto fp32res_fp16_ptr = fp32res_fp16_device.mutable_data<float>(num);
    cast_fp16_to_fp32_cuda_kernel<<<grid, block, 0, stream>>>(
        out_fp16_device.data<float16>(), num, fp32res_fp16_ptr);

    // step4.2 : cast bf16 result to fp32 result
    auto fp32res_bf16_ptr = fp32res_bf16_device.mutable_data<float>(num);
    cast_bf16_to_fp32_cuda_kernel<<<grid, block, 0, stream>>>(
        out_bf16_device.data<bfloat16>(), num, fp32res_bf16_ptr);
  }

  std::vector<float> out_fp32_host(num), out_fp16_host(num), out_bf16_host(num);
  {  // step5 : copy result from device to host
    out_fp32_device.MemcpyToHost(
        out_fp32_host.data(), num * sizeof(float), stream);
    fp32res_fp16_device.MemcpyToHost(
        out_fp16_host.data(), num * sizeof(float), stream);
    fp32res_bf16_device.MemcpyToHost(
        out_bf16_host.data(), num * sizeof(float), stream);
  }

  CUDA_CALL(hipStreamSynchronize(stream));

  for (int i = 0; i < num; ++i) {
    ASSERT_NEAR(out_fp32_host[i], out_fp16_host[i], 1e-2f);
    ASSERT_NEAR(out_fp32_host[i], out_bf16_host[i], 1e-1f);
  }

  CUDA_CALL(hipStreamDestroy(stream));
}

}  // namespace common
}  // namespace cinn
