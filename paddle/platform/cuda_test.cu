#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

#define CHECK_ERR(x)                 \
  if (x != hipSuccess) {            \
    fprintf(stderr,                  \
            "%s in %s at line %d\n", \
            hipGetErrorString(err), \
            __FILE__,                \
            __LINE__);               \
    exit(-1);                        \
  }

__global__ void vecAdd(float *d_A, float *d_B, float *d_C, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    d_C[i] = d_A[i] + d_B[i];
  }
}

TEST(Cuda, Equality) {
  int n = 10;
  // Memory allocation for h_A, h_B and h_C (in the host)
  float h_A[10] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 0.0};
  float h_B[10] = {0.0, 9.0, 8.0, 7.0, 6.0, 5.0, 4.0, 3.0, 2.0, 1.0};
  float h_C[10];
  float *d_A, *d_B, *d_C;
  hipError_t err;
  // Memory allocation for d_A, d_B and d_C (in the device)
  err = hipMalloc((void **)&d_A, sizeof(float) * n);
  CHECK_ERR(err);

  err = hipMalloc((void **)&d_B, sizeof(float) * n);
  CHECK_ERR(err);

  err = hipMalloc((void **)&d_C, sizeof(float) * n);
  CHECK_ERR(err);

  // Copying memory to device
  err = hipMemcpy(d_A, h_A, sizeof(float) * n, hipMemcpyHostToDevice);
  CHECK_ERR(err);

  err = hipMemcpy(d_B, h_B, sizeof(float) * n, hipMemcpyHostToDevice);
  CHECK_ERR(err);

  // Calling the kernel
  vecAdd<<<ceil(n / 256.0), 256>>>(d_A, d_B, d_C, n);

  // Copying results back to host
  err = hipMemcpy(h_C, d_C, sizeof(float) * n, hipMemcpyDeviceToHost);
  CHECK_ERR(err);

  EXPECT_EQ(h_C[0], 1.0);
  for (int i = 1; i < n - 1; ++i) {
    EXPECT_EQ(h_C[i], 11.0);
  }
  EXPECT_EQ(h_C[9], 1.0);
}
