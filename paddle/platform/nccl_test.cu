/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "glog/logging.h"
#include "gtest/gtest.h"
#include "paddle/platform/device_context.h"
#include "paddle/platform/dynload/nccl.h"
#include "paddle/platform/enforce.h"
#include "paddle/platform/gpu_info.h"

#include <thrust/device_vector.h>
#include <memory>
#include <vector>

static int dev_count = 0;

namespace paddle {
namespace platform {

TEST(NCCL, init) {
  std::vector<ncclComm_t> comms;
  comms.resize(dev_count);
  PADDLE_ENFORCE(dynload::ncclCommInitAll(comms.data(), dev_count, nullptr));
  for (int i = 0; i < dev_count; ++i) {
    dynload::ncclCommDestroy(comms[i]);
  }
}

template <typename T>
struct PerThreadData {
  thrust::device_vector<T> send_buff;
  thrust::device_vector<T> recv_buff;
  CUDADeviceContext dev_ctx;

  T* SendBuff() { return thrust::raw_pointer_cast(send_buff.data()); }

  T* RecvBuff() { return thrust::raw_pointer_cast(recv_buff.data()); }

  PerThreadData(int gpu_id, size_t size) : dev_ctx(GPUPlace(gpu_id)) {
    send_buff.resize(size);
    for (size_t i = 0; i < size; ++i) {
      send_buff[i] = static_cast<T>(i);
    }
    recv_buff.resize(size);
  }
};

static constexpr int ELEM_COUNT = 10000;

TEST(NCCL, all_reduce) {
  std::vector<ncclComm_t> comms;
  comms.resize(dev_count);
  VLOG(1) << "Initializing ncclComm";
  PADDLE_ENFORCE(dynload::ncclCommInitAll(comms.data(), dev_count, nullptr));
  VLOG(1) << "ncclComm initialized";
  VLOG(1) << "Creating thread data";
  std::vector<std::unique_ptr<PerThreadData<double>>> data;
  data.reserve(dev_count);
  for (int i = 0; i < dev_count; ++i) {
    VLOG(1) << "Creating thread data for device " << i;
    SetDeviceId(i);
    data.emplace_back(new PerThreadData<double>(i, ELEM_COUNT));
  }
  VLOG(1) << "Thread data created";

  VLOG(1) << "Check send_buf data";
  for (int i = 0; i < dev_count; ++i) {
    VLOG(1) << "Check on device " << i;
    SetDeviceId(i);
    thrust::host_vector<double> tmp = data[i]->send_buff;
    for (size_t j = 0; j < tmp.size(); ++j) {
      ASSERT_NEAR(static_cast<double>(j), tmp[j], 1e-5);
    }
  }

  VLOG(1) << "Invoking ncclAllReduce";

  for (int i = 0; i < dev_count; ++i) {
    VLOG(1) << "Invoking ncclAllReduce with device " << i;
    SetDeviceId(i);
    PADDLE_ENFORCE(dynload::ncclAllReduce(
        data[i]->SendBuff(), data[i]->RecvBuff(), ELEM_COUNT, ncclDouble,
        ncclSum, comms[i], data[i]->dev_ctx.stream()));
    VLOG(1) << "Invoked ncclAllReduce for device " << i;
  }

  VLOG(1) << "Invoked ncclAllReduce";

  VLOG(1) << "Sync devices";
  for (int i = 0; i < dev_count; ++i) {
    VLOG(1) << "Sync device " << i;
    SetDeviceId(i);
    data[i]->dev_ctx.Wait();
  }
  VLOG(1) << "device synced";

  for (int i = 0; i < dev_count; ++i) {
    SetDeviceId(i);
    VLOG(1) << "Checking vector on device " << i;
    thrust::host_vector<double> tmp = data[i]->recv_buff;
    for (size_t j = 0; j < tmp.size(); ++j) {
      auto elem = static_cast<double>(j);
      elem *= dev_count;
      ASSERT_NEAR(tmp[j], elem, 1e-4);
    }
  }

  for (int i = 0; i < dev_count; ++i) {
    dynload::ncclCommDestroy(comms[i]);
  }
}
}  // namespace platform
}  // namespace paddle

int main(int argc, char** argv) {
  // FIXME(tonyyang-svail):
  //   Due to the driver issue on our CI, disable for now
  return 0;
  dev_count = paddle::platform::GetCUDADeviceCount();
  if (dev_count <= 1) {
    LOG(WARNING)
        << "Cannot test multi-gpu nccl, because the CUDA device count is "
        << dev_count;
    return 0;
  }
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
