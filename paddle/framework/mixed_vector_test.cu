#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "gtest/gtest.h"

#include "paddle/framework/init.h"
#include "paddle/framework/mixed_vector.h"

using namespace paddle::framework;
using namespace paddle::platform;
using namespace paddle::memory;

template <typename T>
__global__ void test(T* data, int size) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    data[i] *= 2;
  }
}

TEST(Vector, Normal) {
  // fill the device context pool.
  InitDevices();

  Vector<size_t> vec({1, 2, 3});
  size_t* ptr = vec.data();
  for (size_t i = 0; i < vec.size(); ++i) {
    EXPECT_EQ(vec[i], *(ptr + i));
  }

  vec.clear();
  vec.CopyFromCUDA();

  std::vector<size_t> v = {1, 2, 3};
  for (size_t i = 0; i < v.size(); ++i) {
    EXPECT_EQ(v[i], vec[i]);
  }
}

TEST(Vector, MultipleCopy) {
  InitDevices();
  Vector<size_t> vec({1, 2, 3});
  CUDAPlace place(0);
  vec.mutable_data(place);
  auto vec2 = Vector<size_t>(vec);
  {
    const size_t* ptr = vec2.data(CPUPlace());
    for (size_t i = 0; i < vec2.size(); ++i) {
      EXPECT_EQ(*(ptr + i), vec[i]);
    }
  }
  test<size_t><<<3, 3>>>(vec2.mutable_data(place), vec2.size());
  vec2.CopyFromCUDA();
  {
    const size_t* ptr = vec2.data(CPUPlace());
    for (size_t i = 0; i < vec2.size(); ++i) {
      EXPECT_EQ(*(ptr + i), vec[i] * 2);
    }
  }
}
