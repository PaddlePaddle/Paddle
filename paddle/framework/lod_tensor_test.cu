#include "hip/hip_runtime.h"
/*
  Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.
  Licensed under the Apache License, Version 2.0 (the "License");
  you may not use this file except in compliance with the License.
  You may obtain a copy of the License at
  http://www.apache.org/licenses/LICENSE-2.0
  Unless required by applicable law or agreed to in writing, software
  distributed under the License is distributed on an "AS IS" BASIS,
  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
  See the License for the specific language governing permissions and
  limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "paddle/framework/lod_tensor.h"
#include "paddle/platform/assert.h"

#include <gtest/gtest.h>

__global__ void test(size_t* a, int size) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    a[i] *= 2;
  }
}

TEST(LoDTensor, LoDInGPU) {
  paddle::framework::LoDTensor lod_tensor;
  paddle::platform::GPUPlace place(0);

  paddle::framework::LoD src_lod;
  src_lod.push_back(std::vector<size_t>{0, 2, 4, 6, 8, 10, 12, 14});

  lod_tensor.Resize({14, 16});
  lod_tensor.mutable_data<float>(place);

  lod_tensor.set_lod(src_lod);
  CHECK_EQ(lod_tensor.lod_element(0, 2), 4UL);
  CHECK_EQ(lod_tensor.lod_element(0, 4), 8UL);

  auto lod = lod_tensor.lod();

  test<<<1, 8>>>(lod[0].data(), lod[0].size());
  hipDeviceSynchronize();

  for (size_t i = 0; i < src_lod[0].size(); ++i) {
    CHECK_EQ(lod[0].data()[i], src_lod[0].data()[i] * 2);
  }
}
