#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "glog/logging.h"
#include "paddle/infrt/backends/tensorrt/plugin/plugin_utils.h"
#include "paddle/infrt/backends/tensorrt/plugin/pool_op_plugin.h"
#include "paddle/phi/kernels/funcs/pooling.h"

namespace infrt {
namespace backends {
namespace tensorrt {
namespace plugin {

PoolPlugin::PoolPlugin(bool ceil_mode,
                       PoolType pool_type,
                       bool adaptive,
                       bool exclusive,
                       std::vector<int> ksize,
                       std::vector<int> strides,
                       std::vector<int> paddings,
                       std::vector<int> input_shape,
                       std::vector<int> real_paddings)
    : ceil_mode_(ceil_mode),
      pool_type_(pool_type),
      adaptive_(adaptive),
      exclusive_(exclusive),
      ksize_(ksize),
      strides_(strides),
      paddings_(paddings),
      real_paddings_(real_paddings),
      input_shape_(input_shape) {
  output_shape_ = input_shape_;
  std::vector<int> output_shape =
      CalcOutputSize({input_shape_[1], input_shape_[2]},
                     ceil_mode_,
                     adaptive_,
                     ksize_,
                     strides_,
                     real_paddings_);
  output_shape_[1] = output_shape[0];
  output_shape_[2] = output_shape[1];
}

PoolPlugin::PoolPlugin(void const* serialData, size_t serialLength) {
  // deserializeBase(serialData, serialLength);
  DeserializeValue(&serialData, &serialLength, &ceil_mode_);
  DeserializeValue(&serialData, &serialLength, &pool_type_);
  DeserializeValue(&serialData, &serialLength, &adaptive_);
  DeserializeValue(&serialData, &serialLength, &exclusive_);
  DeserializeValue(&serialData, &serialLength, &ksize_);
  DeserializeValue(&serialData, &serialLength, &strides_);
  DeserializeValue(&serialData, &serialLength, &paddings_);
  DeserializeValue(&serialData, &serialLength, &real_paddings_);
  DeserializeValue(&serialData, &serialLength, &input_shape_);
  DeserializeValue(&serialData, &serialLength, &output_shape_);
}

const char* PoolPlugin::getPluginType() const noexcept { return "pool_plugin"; }

const char* PoolPlugin::getPluginVersion() const noexcept { return "1"; }

int PoolPlugin::getNbOutputs() const noexcept { return 1; }

nvinfer1::Dims PoolPlugin::getOutputDimensions(int outputIndex,
                                               const nvinfer1::Dims* inputs,
                                               int nbInputs) noexcept {
  assert(nbInputs == 1);
  assert(index == 0);
  assert(inputs[0].nbDims == 3);
  nvinfer1::Dims const& input_dims = inputs[0];

  nvinfer1::Dims output_dims = input_dims;

  output_dims.d[1] = output_shape_[1];
  output_dims.d[2] = output_shape_[2];
  return output_dims;
}

int32_t PoolPlugin::initialize() noexcept { return 0; }

void PoolPlugin::terminate() noexcept {}

size_t PoolPlugin::getWorkspaceSize(int32_t maxBatchSize) const noexcept {
  return 0;
}

#if IS_TRT_VERSION_LT(8000)
int PoolPlugin::enqueue(int batch_size,
                        const void* const* inputs,
                        void** outputs,
#else
int PoolPlugin::enqueue(int batch_size,
                        const void* const* inputs,
                        void* const* outputs,
#endif
                        void* workspace,
                        hipStream_t stream) noexcept {
  // TODO(wilber)
  int input_size = 0;
  float const* idata = reinterpret_cast<float const*>(inputs[0]);
  float* const* odatas = reinterpret_cast<float* const*>(outputs);

  std::vector<int> input_shape = input_shape_;
  std::vector<int> output_shape = output_shape_;
  input_shape.insert(input_shape.begin(), batch_size);
  output_shape.insert(output_shape.begin(), batch_size);

  if (pool_type_ == PoolType::max) {
    ::phi::funcs::MaxPool<float> pool_process;
    ::phi::funcs::Pool2dDirectCUDAFunctor<phi::funcs::MaxPool<float>, float>
        pool2d_forward;
    pool2d_forward(idata,
                   input_shape,
                   output_shape,
                   ksize_,
                   strides_,
                   paddings_,
                   true,
                   false,
                   odatas[0],
                   stream,
                   pool_process);
  } else if (pool_type_ == PoolType::avg) {
    ::phi::funcs::AvgPool<float> pool_process;
    ::phi::funcs::Pool2dDirectCUDAFunctor<phi::funcs::AvgPool<float>, float>
        pool2d_forward;
    pool2d_forward(idata,
                   input_shape,
                   output_shape,
                   ksize_,
                   strides_,
                   paddings_,
                   exclusive_,
                   adaptive_,
                   odatas[0],
                   stream,
                   pool_process);
  }

  return hipGetLastError() != hipSuccess;
}

// TODO(wilber): serialize base info?
size_t PoolPlugin::getSerializationSize() const noexcept {
  return SerializedSize(ceil_mode_) + SerializedSize(pool_type_) +
         SerializedSize(adaptive_) + SerializedSize(exclusive_) +
         SerializedSize(ksize_) + SerializedSize(strides_) +
         SerializedSize(paddings_) + SerializedSize(real_paddings_) +
         SerializedSize(input_shape_) + SerializedSize(output_shape_);
}
// TODO(wilber): serialize base info?
void PoolPlugin::serialize(void* buffer) const noexcept {
  // serializeBase(buffer);
  SerializeValue(&buffer, ceil_mode_);
  SerializeValue(&buffer, pool_type_);
  SerializeValue(&buffer, adaptive_);
  SerializeValue(&buffer, exclusive_);
  SerializeValue(&buffer, ksize_);
  SerializeValue(&buffer, strides_);
  SerializeValue(&buffer, paddings_);
  SerializeValue(&buffer, real_paddings_);
  SerializeValue(&buffer, input_shape_);
  SerializeValue(&buffer, output_shape_);
}

void PoolPlugin::destroy() noexcept { delete this; }

void PoolPlugin::setPluginNamespace(char const* plugin_namespace) noexcept {
  namespace_ = plugin_namespace;
}

char const* PoolPlugin::getPluginNamespace() const noexcept {
  return namespace_.c_str();
}

nvinfer1::DataType PoolPlugin::getOutputDataType(
    int32_t index,
    nvinfer1::DataType const* input_types,
    int32_t nbInputs) const noexcept {
  CHECK_EQ(index, 0);
  CHECK_EQ((input_types[0] == nvinfer1::DataType::kFLOAT), true);
  return input_types[0];
}

bool PoolPlugin::isOutputBroadcastAcrossBatch(int32_t outputIndex,
                                              bool const* inputIsBroadcasted,
                                              int32_t nbInputs) const noexcept {
  return false;
}

bool PoolPlugin::canBroadcastInputAcrossBatch(
    int32_t inputIndex) const noexcept {
  return false;
}

nvinfer1::IPluginV2Ext* PoolPlugin::clone() const noexcept {
  auto* plugin = new PoolPlugin(ceil_mode_,
                                pool_type_,
                                adaptive_,
                                exclusive_,
                                ksize_,
                                strides_,
                                paddings_,
                                input_shape_,
                                real_paddings_);
  plugin->setPluginNamespace(namespace_.c_str());
  return plugin;
}

void PoolPlugin::configurePlugin(nvinfer1::PluginTensorDesc const* in,
                                 int32_t nb_input,
                                 nvinfer1::PluginTensorDesc const* out,
                                 int32_t nb_output) noexcept {
  CHECK_EQ(nb_input, 1);
  CHECK_EQ(nb_output, 1);

  input_dims_ = in[0].dims;
  data_format_ = in[0].format;
  data_type_ = in[0].type;
}

bool PoolPlugin::supportsFormatCombination(
    int32_t pos,
    nvinfer1::PluginTensorDesc const* in_out,
    int32_t nb_inputs,
    int32_t nb_outputs) const noexcept {
  CHECK_LT(pos, nb_inputs + nb_outputs);
  CHECK_NOTNULL(in_out);

  return ((in_out[pos].type == nvinfer1::DataType::kFLOAT) &&
          in_out[pos].format == nvinfer1::PluginFormat::kLINEAR);
}

nvinfer1::IPluginV2* PoolPluginCreator::createPlugin(
    const char* name, const nvinfer1::PluginFieldCollection* fc) noexcept {
  // auto* plugin = new UffPoolPluginV2(*fc);
  field_collection_ = *fc;
  plugin_name_ = name;
  const nvinfer1::PluginField* fields = fc->fields;

  bool ceil_mode;
  PoolPlugin::PoolType pool_type;
  bool adaptive;
  bool exclusive;
  std::vector<int> ksize;
  std::vector<int> strides;
  std::vector<int> paddings;
  std::vector<int> real_paddings;
  std::vector<int> input_shape;
  std::vector<int> output_shape;

  // TODO(wilber): add implement.
  CHECK(false) << "not implement";
  // for (int i = 0; i < fc->nbFields; ++i) {
  //   const char* attr_name = fields[i].name;
  //   if (!strcmp(attr_name, "ceil_mode")) {
  //     CHECK_EQ(fields[i].type == nvinfer1::PluginFieldType::kINT8, true);
  //     ceil_mode = *static_cast<const bool*>(fields[i].data);
  //     // mParam.numOutputBoxesPerClass =
  //     //     *(static_cast<const int*>(fields[i].data));
  //   }
  // }

  return nullptr;
}

nvinfer1::IPluginV2* PoolPluginCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength) noexcept {
  auto* plugin = new PoolPlugin(serialData, serialLength);
  plugin_name_ = name;
  return plugin;
}

}  // namespace plugin
}  // namespace tensorrt
}  // namespace backends
}  // namespace infrt
