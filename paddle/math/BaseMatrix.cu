#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <cmath>
#include <string.h>
#include <paddle/utils/Logging.h>
#include "BaseMatrix.h"
#include "hl_matrix_ops.cuh"
#include "hl_matrix_base.cuh"
#include "hl_matrix_apply.cuh"
#include "SIMDFunctions.h"
#include "MathFunctions.h"

namespace paddle {

const char* SPARSE_SUPPORT_ERROR = "Sparse Matrix/Vector is not supported.";

template<class T>
template <class Op>
int BaseMatrixT<T>::applyUnary(Op op) {
  MatrixOffset offset(0, 0);
  applyUnary(op, height_, width_, offset);
  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyUnary(Op op, int numRows, int numCols,
                               MatrixOffset& offset) {
  CHECK(!this->isSparse()) << SPARSE_SUPPORT_ERROR;
  int dimM = numRows;
  int dimN = numCols;
  int lda = stride_;

  T* A = data_;
  CAL_MATRIX_START_ADDRESS(A, height_, width_, lda, offset.aCol_, offset.aRow_);

  CHECK_LE(dimM + offset.aRow_, this->height_);
  CHECK_LE(dimN + offset.aCol_, this->width_);
  if (true == useGpu_) {
    hl_gpu_apply_unary_op(op, A, dimM, dimN, lda);
  } else {
    hl_cpu_apply_unary_op(op, A, dimM, dimN, lda);
  }
  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyBinary(Op op, BaseMatrixT& b) {
  CHECK(height_ == b.height_ && width_ == b.width_)
      << "Matrix dimensions are not equal";

  MatrixOffset offset(0, 0, 0, 0);
  applyBinary(op, b, height_, width_, offset);
  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyBinary(Op op, BaseMatrixT& b, int numRows, int numCols,
                                MatrixOffset& offset) {
  applyBinary(op, b, numRows, numCols, offset, false_type(), false_type());
  return 0;
}

template<class T>
template <class Op, class bAsRowVector, class bAsColVector>
int BaseMatrixT<T>::applyBinary(Op op, BaseMatrixT& b, int numRows, int numCols,
                            MatrixOffset& offset, bAsRowVector, bAsColVector) {
  CHECK(!this->isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(!b.isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(useGpu_ == b.useGpu_) << "Matrix type mismatch";

  int dimM = numRows;
  int dimN = numCols;
  int lda = stride_;
  int ldb = b.stride_;

  T* A = data_;
  T* B = b.data_;
  CAL_MATRIX_START_ADDRESS(A, height_, width_, lda, offset.aCol_, offset.aRow_);
  CAL_MATRIX_START_ADDRESS(B, b.height_, b.width_, ldb, offset.bCol_,
                           offset.bRow_);
  CHECK_LE(dimM + offset.aRow_, this->height_);
  CHECK_LE(dimN + offset.aCol_, this->width_);
  if (!bAsRowVector::value && !bAsColVector::value) {
    CHECK_LE(dimM + offset.bRow_, b.height_);
    CHECK_LE(dimN + offset.bCol_, b.width_);
  } else if (bAsRowVector::value && !bAsColVector::value) {
    CHECK_LE(dimN + offset.bCol_, b.width_);
  } else if (!bAsRowVector::value && bAsColVector::value) {
    CHECK_LE(dimM + offset.bRow_, b.height_);
  } else {
  }
  if (true == useGpu_) {
    hl_gpu_apply_binary_op<T, Op, bAsRowVector::value, bAsColVector::value>(
        op, A, B, dimM, dimN, lda, ldb);
  } else {
    hl_cpu_apply_binary_op<T, Op, bAsRowVector::value, bAsColVector::value>(
        op, A, B, dimM, dimN, lda, ldb);
  }

  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyTernary(Op op, BaseMatrixT& b, BaseMatrixT& c) {
  CHECK_EQ(height_, b.height_);
  CHECK_EQ(width_, b.width_);
  CHECK_EQ(height_, c.height_);
  CHECK_EQ(width_, c.width_);

  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  applyTernary(op, b, c, height_, width_, offset);

  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyTernary(Op op, BaseMatrixT& b, BaseMatrixT& c,
                                 int numRows, int numCols,
                                 MatrixOffset& offset) {
  applyTernary(op, b, c, numRows, numCols, offset, false_type(), false_type());

  return 0;
}

template<class T>
template <class Op, class cAsRowVector, class cAsColVector>
int BaseMatrixT<T>::applyTernary(Op op, BaseMatrixT& b, BaseMatrixT& c,
                                 int numRows, int numCols, MatrixOffset& offset,
                                 cAsRowVector, cAsColVector) {
  CHECK(!this->isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(!b.isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(!c.isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK_EQ(useGpu_, b.useGpu_);
  CHECK_EQ(useGpu_, c.useGpu_);

  int dimM = numRows;
  int dimN = numCols;
  int lda = stride_;
  int ldb = b.stride_;
  int ldc = c.stride_;

  T* A = data_;
  T* B = b.data_;
  T* C = c.data_;
  CAL_MATRIX_START_ADDRESS(A, height_, width_, lda, offset.aCol_, offset.aRow_);
  CAL_MATRIX_START_ADDRESS(B, b.height_, b.width_, ldb, offset.bCol_,
                           offset.bRow_);
  CAL_MATRIX_START_ADDRESS(C, c.height_, c.width_, ldc, offset.cCol_,
                           offset.cRow_);

  CHECK_LE(dimM + offset.aRow_, this->height_);
  CHECK_LE(dimN + offset.aCol_, this->width_);
  CHECK_LE(dimM + offset.bRow_, b.height_);
  CHECK_LE(dimN + offset.bCol_, b.width_);
  if (!cAsRowVector::value && !cAsColVector::value) {
    CHECK_LE(dimM + offset.cRow_, c.height_);
    CHECK_LE(dimN + offset.cCol_, c.width_);
  } else if (cAsRowVector::value && !cAsColVector::value) {
    CHECK_LE(dimN + offset.cCol_, c.width_);
  } else if (!cAsRowVector::value && cAsColVector::value) {
    CHECK_LE(dimM + offset.cRow_, c.height_);
  } else {
  }

  if (true == useGpu_) {
    hl_gpu_apply_ternary_op
      <T, Op, cAsRowVector::value, cAsColVector::value>(
        op, A, B, C, dimM, dimN, lda, ldb, ldc);
  } else {
    hl_cpu_apply_ternary_op
      <T, Op, cAsRowVector::value, cAsColVector::value>(
        op, A, B, C, dimM, dimN, lda, ldb, ldc);
  }

  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyQuaternary(Op op, BaseMatrixT& b, BaseMatrixT& c,
                                    BaseMatrixT& d) {
  CHECK_EQ(height_, b.height_);
  CHECK_EQ(width_, b.width_);
  CHECK_EQ(height_, c.height_);
  CHECK_EQ(width_, c.width_);
  CHECK_EQ(height_, d.height_);
  CHECK_EQ(width_, d.width_);

  MatrixOffset offset(0, 0, 0, 0, 0, 0, 0, 0);
  applyQuaternary(op, b, c, d, height_, width_, offset);

  return 0;
}

template<class T>
template <class Op>
int BaseMatrixT<T>::applyQuaternary(Op op, BaseMatrixT& b, BaseMatrixT& c,
                                    BaseMatrixT& d, int numRows, int numCols,
                                    MatrixOffset& offset) {
  CHECK(!this->isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(!b.isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(!c.isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK(!d.isSparse()) << SPARSE_SUPPORT_ERROR;
  CHECK_EQ(useGpu_, b.useGpu_);
  CHECK_EQ(useGpu_, c.useGpu_);
  CHECK_EQ(useGpu_, d.useGpu_);

  int dimM = numRows;
  int dimN = numCols;
  int lda = stride_;
  int ldb = b.stride_;
  int ldc = c.stride_;
  int ldd = d.stride_;

  T* A = data_;
  T* B = b.data_;
  T* C = c.data_;
  T* D = d.data_;
  CAL_MATRIX_START_ADDRESS(A, height_, width_, lda, offset.aCol_, offset.aRow_);
  CAL_MATRIX_START_ADDRESS(B, b.height_, b.width_, ldb, offset.bCol_,
                           offset.bRow_);
  CAL_MATRIX_START_ADDRESS(C, c.height_, c.width_, ldc, offset.cCol_,
                           offset.cRow_);
  CAL_MATRIX_START_ADDRESS(D, d.height_, d.width_, ldd, offset.dCol_,
                           offset.dRow_);

  CHECK_LE(dimM + offset.aRow_, this->height_);
  CHECK_LE(dimN + offset.aCol_, this->width_);
  CHECK_LE(dimM + offset.bRow_, b.height_);
  CHECK_LE(dimN + offset.bCol_, b.width_);
  CHECK_LE(dimM + offset.cRow_, c.height_);
  CHECK_LE(dimN + offset.cCol_, c.width_);
  CHECK_LE(dimM + offset.dRow_, d.height_);
  CHECK_LE(dimN + offset.dCol_, d.width_);
  if (true == useGpu_) {
    hl_gpu_apply_quaternary_op(op, A, B, C, D, dimM, dimN, lda, ldb,
                               ldc, ldd);
  } else {
    hl_cpu_apply_quaternary_op(op, A, B, C, D, dimM, dimN, lda, ldb,
                               ldc, ldd);
  }

  return 0;
}

template<class T>
template <class Agg, class Op, class Saver, class aAsRowVector,
          class aAsColVector>
int BaseMatrixT<T>::aggregate(Agg agg, Op op, Saver sv, BaseMatrixT& b,
                              int numRows, int numCols, MatrixOffset& offset,
                              aAsRowVector, aAsColVector) {
  CHECK_EQ(useGpu_, b.useGpu_);

  int ld = stride_;
  int ldb = b.stride_;

  T* dst = data_;
  T* B = b.data_;
  CAL_MATRIX_START_ADDRESS(dst, height_, width_, ld, offset.aCol_,
                           offset.aRow_);
  CAL_MATRIX_START_ADDRESS(B, b.height_, b.width_, ldb, offset.bCol_,
                           offset.bRow_);

  if (aAsRowVector::value && !aAsColVector::value) {
    if (useGpu_) {
      hl_gpu_matrix_column_op(agg, op, sv, numRows, numCols, dst, B, ldb);
    } else {
      hl_cpu_matrix_column_op(agg, op, sv, numRows, numCols, dst, B, ldb);
    }
  } else if (!aAsRowVector::value && aAsColVector::value) {
    if (useGpu_) {
      hl_gpu_matrix_row_op(agg, op, sv, numRows, numCols, dst, ld, B, ldb);
    } else {
      hl_cpu_matrix_row_op(agg, op, sv, numRows, numCols, dst, ld, B, ldb);
    }
  } else {
    LOG(FATAL) << "not supported";
  }

  return 0;
}

template<class T>
template <class Agg, class Op, class Saver, class aAsRowVector,
          class aAsColVector>
int BaseMatrixT<T>::aggregate(Agg agg, Op op, Saver sv, BaseMatrixT& b,
                              BaseMatrixT& c, int numRows, int numCols,
                              MatrixOffset& offset, aAsRowVector,
                              aAsColVector) {
  CHECK_EQ(useGpu_, b.useGpu_);
  CHECK_EQ(useGpu_, c.useGpu_);

  int ld = stride_;
  int ldb = b.stride_;
  int ldc = c.stride_;

  T* dst = data_;
  T* B = b.data_;
  T* C = c.data_;
  CAL_MATRIX_START_ADDRESS(dst, height_, width_, ld, offset.aCol_,
                           offset.aRow_);
  CAL_MATRIX_START_ADDRESS(B, b.height_, b.width_, ldb, offset.bCol_,
                           offset.bRow_);
  CAL_MATRIX_START_ADDRESS(C, c.height_, c.width_, ldc, offset.cCol_,
                           offset.cRow_);

  if (aAsRowVector::value && !aAsColVector::value) {
    if (useGpu_) {
      hl_gpu_matrix_column_op(agg, op, sv, numRows, numCols, dst, B,
                              ldb, C, ldc);
    } else {
      hl_cpu_matrix_column_op(agg, op, sv, numRows, numCols, dst, B,
                              ldb, C, ldc);
    }
  } else if (!aAsRowVector::value && aAsColVector::value) {
    if (useGpu_) {
      hl_gpu_matrix_row_op(agg, op, sv, numRows, numCols, dst, ld, B,
                           ldb, C, ldc);
    } else {
      hl_cpu_matrix_row_op(agg, op, sv, numRows, numCols, dst, ld, B,
                           ldb, C, ldc);
    }
  } else {
    LOG(FATAL) << "not supported";
  }

  return 0;
}

/**
 * @brief   unary operator.
 *
 */

DEFINE_MATRIX_UNARY_OP(Neg, a = -a);
template<class T>
void BaseMatrixT<T>::neg() { applyUnary(unary::Neg<T>()); }

DEFINE_MATRIX_UNARY_OP(Exp, a = exp(a));
template<>
void BaseMatrixT<real>::exp2() { applyUnary(unary::Exp<real>()); }

DEFINE_MATRIX_UNARY_OP(Log, a = log(a));
template<>
void BaseMatrixT<real>::log2() {
  if (useGpu_) {
    applyUnary(unary::Log<real>());
  } else {
    vLog(height_ * width_, data_, data_);
  }
}

DEFINE_MATRIX_UNARY_OP(Sqrt, a = sqrt(a));
template<>
void BaseMatrixT<real>::sqrt2() { applyUnary(unary::Sqrt<real>()); }

DEFINE_MATRIX_UNARY_OP(Square, a = a * a);
template<class T>
void BaseMatrixT<T>::square2() { applyUnary(unary::Square<T>()); }

DEFINE_MATRIX_UNARY_OP(Reciprocal, a = 1.0f / a);
template<class T>
void BaseMatrixT<T>::reciprocal2() { applyUnary(unary::Reciprocal<T>()); }

DEFINE_MATRIX_UNARY_OP(Abs, a = a > 0 ? a : -a);
template<class T>
void BaseMatrixT<T>::abs2() { applyUnary(unary::Abs<T>()); }

DEFINE_MATRIX_UNARY_OP(Sign, a = (a > 0) - (a < 0));
template<class T>
void BaseMatrixT<T>::sign2() { applyUnary(unary::Sign<T>()); }

DEFINE_MATRIX_UNARY_OP(Zero, a = 0);
template<class T>
void BaseMatrixT<T>::zero() { applyUnary(unary::Zero<T>()); }

template<class T>
void BaseMatrixT<T>::zeroAtOffset(int64_t columnOffset, int64_t numColumns) {
  int numRows = height_;
  int numCols = numColumns;
  MatrixOffset offset(columnOffset, 0);
  applyUnary(unary::Zero<T>(), numRows, numCols, offset);
}

DEFINE_MATRIX_UNARY_OP(One, a = 1);
template<class T>
void BaseMatrixT<T>::one() { applyUnary(unary::One<T>()); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(Pow, ONE_PARAMETER, a = pow(a, p));
template<>
void BaseMatrixT<real>::pow2(real p) {
  if (useGpu_) {
    applyUnary(unary::Pow<real>(p));
  } else {
    vPow(height_ * width_, data_, p, data_);
  }
}

DEFINE_MATRIX_UNARY_PARAMETER_OP(SubScalar, ONE_PARAMETER, a -= p);
template<class T>
void BaseMatrixT<T>::subScalar(T p) { applyUnary(unary::SubScalar<T>(p)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(MulScalar, ONE_PARAMETER, a *= p);
template<class T>
void BaseMatrixT<T>::mulScalar(T p) { applyUnary(unary::MulScalar<T>(p)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(DivScalar, ONE_PARAMETER, a /= p);
template<class T>
void BaseMatrixT<T>::divScalar(T p) { applyUnary(unary::DivScalar<T>(p)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(Assign, ONE_PARAMETER, a = p);
template<class T>
void BaseMatrixT<T>::assign(T p) { applyUnary(unary::Assign<T>(p)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(Add, ONE_PARAMETER, a += p);
template<class T>
void BaseMatrixT<T>::add(T p) { applyUnary(unary::Add<T>(p)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(Add2, TWO_PARAMETER, a = a * p1 + p2);
template<class T>
void BaseMatrixT<T>::add(T p1, T p2) { applyUnary(unary::Add2<T>(p1, p2)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(Clip, TWO_PARAMETER,
                                 a = a < p1 ? p1 : (a > p2 ? p2 : a));
template<class T>
void BaseMatrixT<T>::clip(T p1, T p2) { applyUnary(unary::Clip<T>(p1, p2)); }

DEFINE_MATRIX_UNARY_PARAMETER_OP(BiggerThanScalar, ONE_PARAMETER,
                                 a = a > p ? 1.0f : 0.0f);
template<class T>
void BaseMatrixT<T>::biggerThanScalar(T p) {
  applyUnary(unary::BiggerThanScalar<T>(p));
}

DEFINE_MATRIX_UNARY_PARAMETER_OP(DownClip, ONE_PARAMETER,
                                 a = a > p ? a : p);
template<class T>
void BaseMatrixT<T>::downClip(T p) {
  applyUnary(unary::DownClip<T>(p));
}

/**
 * @brief   binary operator.
 *
 */

DEFINE_MATRIX_BINARY_OP(Add, a += b);
template<class T>
void BaseMatrixT<T>::add(BaseMatrixT& b) {
  applyBinary(binary::Add<T>(), b);
}

template<>
void BaseMatrixT<real>::add(BaseMatrixT& b) {
  if (useGpu_) {
    applyBinary(binary::Add<real>(), b);
  } else {  // cpu branch
    CHECK_EQ(height_, b.height_);
    CHECK_EQ(width_, b.width_);
    vAdd(height_ * width_, data_, b.data_, data_);
  }
}

template<class T>
void BaseMatrixT<T>::addAtOffset(BaseMatrixT& b, int64_t columnOffset) {
  if (columnOffset + b.width_ <= width_) {
    int numRows = height_;
    int numCols = b.width_;
    MatrixOffset offset(columnOffset, 0, 0, 0);
    applyBinary(binary::Add<T>(), b, numRows, numCols, offset);
  } else if (columnOffset + width_ <= b.width_) {
    int numRows = height_;
    int numCols = width_;
    MatrixOffset offset(0, 0, columnOffset, 0);
    applyBinary(binary::Add<T>(), b, numRows, numCols, offset);
  } else {
    LOG(FATAL) << "Wrong argument "
               << " a.width=" << width_ << " b.width=" << b.width_
               << " columnOffset=" << columnOffset;
  }
}

template<class T>
void BaseMatrixT<T>::addP2P(BaseMatrixT& b) {
  T* A = data_;
  T* B = b.data_;
  int dimM = height_;
  int dimN = width_;

  hl_gpu_apply_binary_op<T, binary::Add<T>, 0, 0>
    (binary::Add<T>(), A, B, dimM, dimN, dimN, dimN);
}

template<class T>
void BaseMatrixT<T>::addColVector(BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0);
  int numRows = height_;
  int numCols = width_;
  applyBinary(binary::Add<T>(), b, numRows, numCols, offset, false_type(),
              true_type() /* bAsColVector */);
}

template<class T>
void BaseMatrixT<T>::addRowVector(BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0);
  int numRows = height_;
  int numCols = width_;
  applyBinary(binary::Add<T>(), b, numRows, numCols, offset,
              true_type() /* bAsRowVector */, false_type());
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(Add1, ONE_PARAMETER, a += b * p);
template<class T>
void BaseMatrixT<T>::add(BaseMatrixT& b, T p) {
  applyBinary(binary::Add1<T>(p), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(Pow, ONE_PARAMETER, a = pow(b, p));
template<>
void BaseMatrixT<real>::pow2(BaseMatrixT& b, real p) {
  if (useGpu_) {
    applyBinary(binary::Pow<real>(p), b);
  } else {
    vPow(height_ * width_, b.data_, p, data_);
  }
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(Add2, TWO_PARAMETER, a = p1 * a + p2 * b);
template<class T>
void BaseMatrixT<T>::add(BaseMatrixT& b, T p1, T p2) {
  applyBinary(binary::Add2<T>(p1, p2), b);
}

template<class T>
void BaseMatrixT<T>::addBias(BaseMatrixT& b, T scale) {
  MatrixOffset offset(0, 0, 0, 0);
  int numRows = height_;
  int numCols = width_;
  applyBinary(binary::Add1<T>(scale), b, numRows, numCols, offset,
              true_type() /* bAsRowVector */, false_type());
}

DEFINE_MATRIX_BINARY_OP(Sub, a -= b);
template<class T>
void BaseMatrixT<T>::sub(BaseMatrixT& b) { applyBinary(binary::Sub<T>(), b); }

DEFINE_MATRIX_BINARY_PARAMETER_OP(Sub1, ONE_PARAMETER, a -= b * p);
template<class T>
void BaseMatrixT<T>::sub(BaseMatrixT& b, T p) {
  applyBinary(binary::Sub1<T>(p), b);
}

DEFINE_MATRIX_BINARY_OP(Relu, b = a > 0.0f ? a : 0.0f);
template<class T>
void BaseMatrixT<T>::relu(BaseMatrixT& b) { applyBinary(binary::Relu<T>(), b); }

DEFINE_MATRIX_BINARY_OP(ReluDerivative, a *= (b > 0.0f ? 1.0f : 0.0f));
template<class T>
void BaseMatrixT<T>::reluDerivative(BaseMatrixT& b) {
  applyBinary(binary::ReluDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(Softrelu, const T THRESHOLD = 40.0;
                        b = log(1.0 + exp((a > THRESHOLD)
                                              ? THRESHOLD
                                              : ((a < -THRESHOLD) ? (-THRESHOLD)
                                                                  : a))));
template<>
void BaseMatrixT<real>::softrelu(BaseMatrixT& b) {
  applyBinary(binary::Softrelu<real>(), b);
}

DEFINE_MATRIX_BINARY_OP(
    SoftreluDerivative, const T THRESHOLD = 40.0;
    a *= (1.0 - exp(-1.0 * ((b > THRESHOLD)
                                ? THRESHOLD
                                : ((b < -THRESHOLD) ? (-THRESHOLD) : b)))));
template<>
void BaseMatrixT<real>::softreluDerivative(BaseMatrixT& b) {
  applyBinary(binary::SoftreluDerivative<real>(), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(Brelu, TWO_PARAMETER, b = a > p1 ? a : p1;
                                  b = b < p2 ? b : p2);
template<class T>
void BaseMatrixT<T>::brelu(BaseMatrixT& b) {
  int p1 = 0, p2 = 24;    //! TODO(yuyang18): Make p1,p2 configuable.
  applyBinary(binary::Brelu<T>(p1, p2), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(BreluDerivative, TWO_PARAMETER,
                                  a *= (b > p1 && b < p2) ? 1.0 : 0.0);
template<class T>
void BaseMatrixT<T>::breluDerivative(BaseMatrixT& b) {
  int p1 = 0, p2 = 24;
  applyBinary(binary::BreluDerivative<T>(p1, p2), b);
}

DEFINE_MATRIX_BINARY_OP(Square, b = a * a);
template<class T>
void BaseMatrixT<T>::square2(BaseMatrixT& b) {
  applyBinary(binary::Square<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(SquareDerivative, a *= 2.0 * b);
template<class T>
void BaseMatrixT<T>::squareDerivative(BaseMatrixT& b) {
  applyBinary(binary::SquareDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(Tanh,
    T tmp = -2.0 * a;
    tmp = (tmp > EXP_MAX_INPUT) ? EXP_MAX_INPUT : tmp;
    b = 2.0 / (1.0 + std::exp(tmp)) - 1.0);
template<>
void BaseMatrixT<real>::tanh(BaseMatrixT& b) {
  applyBinary(binary::Tanh<real>(), b);
}

DEFINE_MATRIX_BINARY_OP(TanhDerivative, a *= 1 - b * b);
template<class T>
void BaseMatrixT<T>::tanhDerivative(BaseMatrixT& b) {
  applyBinary(binary::TanhDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(ScaledTanh, TWO_PARAMETER,
                                  b = p1 *
                                      (2.0 / (1.0 + exp(-2 * p2 * a)) - 1.0));
template<>
void BaseMatrixT<real>::scaledTanh(BaseMatrixT& b, real p1, real p2) {
  applyBinary(binary::ScaledTanh<real>(p1, p2), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(ScaledTanhDerivative, TWO_PARAMETER,
                                  a *= p2 * (p1 - b * b));
template<class T>
void BaseMatrixT<T>::scaledTanhDerivative(BaseMatrixT& b, T p1, T p2) {
  applyBinary(binary::ScaledTanhDerivative<T>(p1 * p1, p2 / p1), b);
}

DEFINE_MATRIX_BINARY_OP(Reciprocal, b = 1.0f / a);
template<class T>
void BaseMatrixT<T>::reciprocal2(BaseMatrixT& b) {
  applyBinary(binary::Reciprocal<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(ReciprocalDerivative, a *= -b * b);
template<class T>
void BaseMatrixT<T>::reciprocalDerivative(BaseMatrixT& b) {
  applyBinary(binary::ReciprocalDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(Abs, b = a > 0.0f ? a : -a);
template<class T>
void BaseMatrixT<T>::abs2(BaseMatrixT& b) { applyBinary(binary::Abs<T>(), b); }

DEFINE_MATRIX_BINARY_OP(AbsDerivative, a = (b > 0) ? a : (b < 0) ? -a : 0);
template<class T>
void BaseMatrixT<T>::absDerivative(BaseMatrixT& b) {
  applyBinary(binary::AbsDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(
    Sigmoid, const T THRESHOLD_MIN = -40.0; const T THRESHOLD_MAX = 13.0;
    T tmp = (a < THRESHOLD_MIN) ? THRESHOLD_MIN
                                   : ((a > THRESHOLD_MAX) ? THRESHOLD_MAX : a);
    b = 1.0f / (1.0f + exp(-tmp)));
template<>
void BaseMatrixT<real>::sigmoid(BaseMatrixT& b) {
  if (useGpu_) {
    applyBinary(binary::Sigmoid<real>(), b);
  } else {  // cpu versioni
    size_t numSamples = this->height_;
    size_t dim = this->width_;
    CHECK_EQ(b.height_, numSamples);
    CHECK_EQ(b.width_, dim);
    const real* in = this->data_;
    real* out = b.data_;

    // out = - in
    const float THRESHOLD_MIN = -40.0;  // make sure sigmoid(x) > 0
    const float THRESHOLD_MAX = 13.0;   // make sure sigmoid(x) < 1
    for (size_t i = 0; i < numSamples * dim; ++i) {
      real tmp = in[i];
      tmp = (tmp < THRESHOLD_MIN)
                ? THRESHOLD_MIN
                : ((tmp > THRESHOLD_MAX) ? THRESHOLD_MAX : tmp);
      out[i] = -tmp;
    }

    // out = exp(out)
    vExp(numSamples * dim, out, out);

    // out = 1 / (1 + out)
    for (size_t i = 0; i < numSamples * dim; ++i) {
      out[i] = 1 / (1 + out[i]);
    }
  }
}

DEFINE_MATRIX_BINARY_OP(SigmoidDerivative, a *= b * (1 - b));
template<class T>
void BaseMatrixT<T>::sigmoidDerivative(BaseMatrixT& b) {
  applyBinary(binary::SigmoidDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(ExpDerivative, a *= b);
template<class T>
void BaseMatrixT<T>::expDerivative(BaseMatrixT& b) {
  applyBinary(binary::ExpDerivative<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(Sign, b = a > 0.0f ? 1.0f : -1.0f);
template<class T>
void BaseMatrixT<T>::sign2(BaseMatrixT& b) {
  applyBinary(binary::Sign<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(Exp, a = exp(b));
template<>
void BaseMatrixT<real>::exp2(BaseMatrixT& b) {
  applyBinary(binary::Exp<real>(), b);
}

DEFINE_MATRIX_BINARY_OP(Log, a = log(b));
template<>
void BaseMatrixT<real>::log2(BaseMatrixT& b) {
  if (useGpu_) {
    applyBinary(binary::Log<real>(), b);
  } else {
    vLog(height_ * width_, b.data_, data_);
  }
}

DEFINE_MATRIX_BINARY_OP(Sqrt, a = sqrt(b));
template<>
void BaseMatrixT<real>::sqrt2(BaseMatrixT& b) {
  applyBinary(binary::Sqrt<real>(), b);
}

DEFINE_MATRIX_BINARY_OP(InvSqrt, a = 1.0f / sqrt(b));
template<>
void BaseMatrixT<real>::invSqrt(BaseMatrixT& b) {
  if (useGpu_) {
    applyBinary(binary::InvSqrt<real>(), b);
  } else {  // cpu branch
    CHECK_EQ(height_, b.height_);
    CHECK_EQ(width_, b.width_);
    vInvSqrt(height_ * width_, b.data_, data_);
  }
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(IsEqual, ONE_PARAMETER, a = (b == p));
template<class T>
void BaseMatrixT<T>::isEqualTo(BaseMatrixT& b, T value) {
  applyBinary(binary::IsEqual<T>(value), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(AddScalar, ONE_PARAMETER, a = b + p);
template<class T>
void BaseMatrixT<T>::addScalar(BaseMatrixT& b, T p) {
  applyBinary(binary::AddScalar<T>(p), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(SubScalar, ONE_PARAMETER, a = b - p);
template<class T>
void BaseMatrixT<T>::subScalar(BaseMatrixT& b, T p) {
  applyBinary(binary::SubScalar<T>(p), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(MulScalar, ONE_PARAMETER, a = b * p);
template<class T>
void BaseMatrixT<T>::mulScalar(BaseMatrixT& b, T p) {
  applyBinary(binary::MulScalar<T>(p), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(DivScalar, ONE_PARAMETER, a = b / p);
template<class T>
void BaseMatrixT<T>::divScalar(BaseMatrixT& b, T p) {
  applyBinary(binary::DivScalar<T>(p), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(ScalarDiv, ONE_PARAMETER, a = p / b);
template<class T>
void BaseMatrixT<T>::scalarDiv(BaseMatrixT& b, T p) {
  applyBinary(binary::ScalarDiv<T>(p), b);
}

/**
 * @brief   ternary operator.
 *
 */

DEFINE_MATRIX_TERNARY_OP(SoftCrossEntropy,
                         a = -c * log(b) - (1 - c) * log(1 - b));
template<>
void BaseMatrixT<real>::softCrossEntropy(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::SoftCrossEntropy<real>(), b, c);
}

DEFINE_MATRIX_TERNARY_OP(SoftCrossEntropyBp, a += (b - c) / (b * (1 - b)));
template<class T>
void BaseMatrixT<T>::softCrossEntropyBp(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::SoftCrossEntropyBp<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_OP(BinaryCrossEntropy,
                         a = c > 0.5 ? -log(b) : -log(1.0 - b));
template<>
void BaseMatrixT<real>::binaryLabelCrossEntropy(BaseMatrixT& b,
                                                BaseMatrixT& c) {
  if (useGpu_) {
    applyTernary(ternary::BinaryCrossEntropy<real>(), b, c);
  } else {
    CHECK_EQ(height_, b.height_);
    CHECK_EQ(height_, c.height_);
    CHECK_EQ(width_, b.width_);
    CHECK_EQ(width_, c.width_);

    size_t size = height_ * width_;
    real* out = b.data_;
    real* label = c.data_;
    real* cost = data_;

    for (size_t i = 0; i < size; ++i) {
      cost[i] = label[i] > 0.5 ? out[i] : 1.0 - out[i];
    }
    vLog(size, cost, cost);
    for (size_t i = 0; i < size; ++i) {
      cost[i] *= -1.0;
    }
  }
}

DEFINE_MATRIX_TERNARY_OP(BinaryCrossEntropyBp,
                         a += c > 0.5 ? -1.0 / b : 1.0 / (1.0 - b));
template<class T>
void BaseMatrixT<T>::binaryLabelCrossEntropyBp(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::BinaryCrossEntropyBp<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_OP(Add, a = b + c);
template<class T>
void BaseMatrixT<T>::add(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::Add<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(Add1, TWO_PARAMETER, a = p1 * b + p2 * c);
template<class T>
void BaseMatrixT<T>::add(BaseMatrixT& b, T p1, BaseMatrixT& c, T p2) {
  applyTernary(ternary::Add1<T>(p1, p2), b, c);
}

DEFINE_MATRIX_TERNARY_OP(Sub, a = b - c);
template<class T>
void BaseMatrixT<T>::sub(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::Sub<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(Sub1, TWO_PARAMETER, a = p1 * b - p2 * c);
template<class T>
void BaseMatrixT<T>::sub(BaseMatrixT& b, T p1, BaseMatrixT& c, T p2) {
  applyTernary(ternary::Sub1<T>(p1, p2), b, c);
}

DEFINE_MATRIX_TERNARY_OP(Add2, a = a + b + c);
template<class T>
void BaseMatrixT<T>::add2(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::Add2<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(Add3, THREE_PARAMETER,
                                   a = p1 * a + p2 * b + p3 * c);
template<class T>
void BaseMatrixT<T>::add2(BaseMatrixT& b, BaseMatrixT& c, T p1, T p2, T p3) {
  applyTernary(ternary::Add3<T>(p1, p2, p3), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(SgdUpdate, THREE_PARAMETER,
                                   c = p2 * c - p1 * (b + p3 * a);
                                   a = a + c);
template<class T>
void BaseMatrixT<T>::sgdUpdate(BaseMatrixT& b,  // grad
                               BaseMatrixT& c,  // mom
                               T p1,        // learningRate,
                               T p2,        // momentum,
                               T p3) {      // decayRate
  applyTernary(ternary::SgdUpdate<T>(p1, p2, p3), b, c);
}

DEFINE_MATRIX_QUATERNARY_PARAMETER_OP(SgdUpdate, THREE_PARAMETER,
                                      c = p2 * c - p1 * d * (b + p3 * a);
                                      a += c);
template<class T>
void BaseMatrixT<T>::sgdUpdate(BaseMatrixT& b,  // grad,
                               BaseMatrixT& c,  // mom,
                               BaseMatrixT& d,  // lr,
                               T p1,        // learningRate,
                               T p2,        // momentum,
                               T p3) {      // decayRate
  applyQuaternary(quaternary::SgdUpdate<T>(p1, p2, p3), b, c, d);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(ApplyL1, ONE_PARAMETER, T lambda = p * b;
                                  a = (a > lambda)
                                          ? (a - lambda)
                                          : (a < -lambda) ? (a + lambda) : 0);
template<class T>
void BaseMatrixT<T>::applyL1(BaseMatrixT& lr, T learningRate, T decayRate) {
  applyBinary(binary::ApplyL1<T>(learningRate * decayRate), lr);
}

template<>
void BaseMatrixT<real>::applyL1(BaseMatrixT& lr,
                                real learningRate,
                                real decayRate) {
  if (useGpu_) {
    applyBinary(binary::ApplyL1<real>(learningRate * decayRate), lr);
  } else {
    simd::decayL1(this->data_, this->data_, lr.data_, learningRate * decayRate,
                  height_ * width_);
  }
}

DEFINE_MATRIX_UNARY_PARAMETER_OP(ApplyL1, ONE_PARAMETER, T lambda = p;
                                 a = (a > lambda)
                                         ? (a - lambda)
                                         : (a < -lambda) ? (a + lambda) : 0);
template<class T>
void BaseMatrixT<T>::applyL1(T learningRate, T decayRate) {
  applyUnary(unary::ApplyL1<T>(learningRate * decayRate));
}

template<>
void BaseMatrixT<real>::applyL1(real learningRate, real decayRate) {
  if (useGpu_) {
    applyUnary(unary::ApplyL1<real>(learningRate * decayRate));
  } else {
    simd::decayL1(this->data_, this->data_, learningRate * decayRate,
                  height_ * width_);
  }
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(ApplyL2, ONE_PARAMETER,
                                  a *= (1.0f / (1.0f + p * b)));
template<class T>
void BaseMatrixT<T>::applyL2(BaseMatrixT& lr, T learningRate, T decayRate) {
  if (useGpu_) {
    applyBinary(binary::ApplyL2<T>(learningRate * decayRate), lr);
  } else {
    size_t size = this->height_ * this->width_;
    T decay = learningRate * decayRate;
    for (size_t j = 0; j < size; ++j) {
      this->data_[j] *= 1.0f / (1.0f + decay * lr.data_[j]);
    }
  }
}

template<class T>
void BaseMatrixT<T>::applyL2(T learningRate, T decayRate) {
  BaseMatrixT<T>::mulScalar(1.0f / (1.0f + learningRate * decayRate));
}

DEFINE_MATRIX_BINARY_OP(DotMul, a *= b);
template<class T>
void BaseMatrixT<T>::dotMul(BaseMatrixT& b) {
  applyBinary(binary::DotMul<T>(), b);
}

DEFINE_MATRIX_TERNARY_OP(DotMul, a = b * c);
template<class T>
void BaseMatrixT<T>::dotMul(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::DotMul<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_OP(DotDiv, a = (b == 0.0) ? 0.0 : b / c);
template<class T>
void BaseMatrixT<T>::dotDiv(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::DotDiv<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(DotDiv2P, TWO_PARAMETER,
                                   a = (b + p1) / (c + p2));
template<class T>
void BaseMatrixT<T>::dotDiv(BaseMatrixT& b, BaseMatrixT& c, T p1, T p2) {
  applyTernary(ternary::DotDiv2P<T>(p1, p2), b, c);
}

DEFINE_MATRIX_QUATERNARY_OP(RankLoss, const T THRESHOLD = 40.0; a = b - c;
                            a = (a > THRESHOLD)
                                    ? THRESHOLD
                                    : ((a < -THRESHOLD) ? (-THRESHOLD) : a);
                            a = log(1 + exp(a)) - a * d);
template<>
void BaseMatrixT<real>::rankLoss(BaseMatrixT& b,
                                 BaseMatrixT& c,
                                 BaseMatrixT& d) {
  applyQuaternary(quaternary::RankLoss<real>(), b, c, d);
}

DEFINE_MATRIX_QUATERNARY_OP(RankLossBp, const T THRESHOLD = 40.0; a = b - c;
                            a = (a > THRESHOLD)
                                    ? THRESHOLD
                                    : ((a < -THRESHOLD) ? (-THRESHOLD) : a);
                            a = exp(a); a = (a / (1 + a) - d));
template<>
void BaseMatrixT<real>::rankLossBp(BaseMatrixT& b,
                                   BaseMatrixT& c,
                                   BaseMatrixT& d) {
  applyQuaternary(quaternary::RankLossBp<real>(), b, c, d);
}

/* this = log(1 + exp(b)) - c * b */
DEFINE_MATRIX_TERNARY_OP(LogisticRegressionLoss, const T THRESHOLD = 40.0;
                         T x = (b > THRESHOLD) ? THRESHOLD : (b < -THRESHOLD)
                                                                 ? -THRESHOLD
                                                                 : b;
                         a = log(1 + exp(x)) - c * x);
template<>
void BaseMatrixT<real>::logisticRegressionLoss(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::LogisticRegressionLoss<real>(), b, c);
}

/* this = exp(b)/(1+exp(b)) - c */
DEFINE_MATRIX_TERNARY_OP(LogisticRegressionLossBp, const T THRESHOLD = 40.0;
                         T x = (b > THRESHOLD) ? THRESHOLD : (b < -THRESHOLD)
                                                                 ? -THRESHOLD
                                                                 : b;
                         x = exp(x); a = x / (1 + x) - c);
template<>
void BaseMatrixT<real>::logisticRegressionLossBp(BaseMatrixT& b,
                                                 BaseMatrixT& c) {
  applyTernary(ternary::LogisticRegressionLossBp<real>(), b, c);
}

DEFINE_MATRIX_TERNARY_OP(BiggerThan, a = (b > c) ? 1.0f : 0.0f);
template<class T>
void BaseMatrixT<T>::biggerThan(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::BiggerThan<T>(), b, c);
}

DEFINE_MATRIX_QUATERNARY_OP(
    BiggerThan, a = ((b > c && d > 0.5f) || (b < c && d < 0.5f)) ? 1.0f : 0.0f);
template<class T>
void BaseMatrixT<T>::biggerThan(BaseMatrixT& b,
                                BaseMatrixT& c,
                                BaseMatrixT& d) {
  applyQuaternary(quaternary::BiggerThan<T>(), b, c, d);
}

DEFINE_MATRIX_TERNARY_OP(Max, a = (b > c) ? b : c);
template<class T>
void BaseMatrixT<T>::max2(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::Max<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(BinaryClassificationError, ONE_PARAMETER,
                                   c += ((a > p) == (b > p)) ? 0.0f : 1.0f);
template<class T>
void BaseMatrixT<T>::binaryClassificationError2(size_t destCol, BaseMatrixT& b,
                                                BaseMatrixT& c, T p) {
  CHECK(!useGpu_) << "do not support gpu";
  MatrixOffset offset(0, 0, 0, 0, destCol, 0);
  int numRows = b.height_;
  int numCols = b.width_;
  b.applyTernary(ternary::BinaryClassificationError<T>(p), c, *this, numRows,
                 numCols, offset, false_type(), true_type() /*cAsColVector*/);
}

template<>
void BaseMatrixT<real>::binaryClassificationError(size_t destCol,
                                                  BaseMatrixT& b,
                                                  BaseMatrixT& c,
                                                  real p) {
  MatrixOffset offset(destCol, 0, 0, 0, 0, 0);
  int numRows = b.height_;
  int numCols = b.width_;
  aggregate(aggregate::sum(), base::binary::classificationError(p),
            base::binary::add(), b, c, numRows, numCols, offset, false_type(),
            true_type() /*aAsColVector*/);
}

DEFINE_MATRIX_QUATERNARY_PARAMETER_OP(Add3, THREE_PARAMETER,
                                      a = p1 * b + p2 * c + p3 * d);
template<class T>
void BaseMatrixT<T>::add3(BaseMatrixT& b, BaseMatrixT& c, BaseMatrixT& d, T p1,
                          T p2, T p3) {
  applyQuaternary(quaternary::Add3<T>(p1, p2, p3), b, c, d);
}

DEFINE_MATRIX_TERNARY_OP(DotMulSquare, a = b * c * c);
template<class T>
void BaseMatrixT<T>::dotMulSquare(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::DotMulSquare<T>(), b, c);
}

DEFINE_MATRIX_TERNARY_OP(DotSquareSquare, a = b * b * c * c);
template<class T>
void BaseMatrixT<T>::dotSquareSquare(BaseMatrixT& b, BaseMatrixT& c) {
  applyTernary(ternary::DotSquareSquare<T>(), b, c);
}

DEFINE_MATRIX_BINARY_OP(DotMulSquare, a *= b * b);
template<class T>
void BaseMatrixT<T>::dotMulSquare(BaseMatrixT& b) {
  applyBinary(binary::DotMulSquare<T>(), b);
}

DEFINE_MATRIX_BINARY_OP(DotSquareMul, a = a * a * b);
template<class T>
void BaseMatrixT<T>::dotSquareMul(BaseMatrixT& b) {
  applyBinary(binary::DotSquareMul<T>(), b);
}

DEFINE_MATRIX_QUATERNARY_PARAMETER_OP(AddSquareSum, THREE_PARAMETER,
                                      T tmp = p1 * b + p2 * c + p3 * d;
                                      a += tmp * tmp);
template<class T>
void BaseMatrixT<T>::addSquareSum(BaseMatrixT& b, BaseMatrixT& c, BaseMatrixT d,
                                  T p1, T p2, T p3) {
  applyQuaternary(quaternary::AddSquareSum<T>(p1, p2, p3), b, c, d);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(AddSquare, ONE_PARAMETER, a += p * b * b);
template<class T>
void BaseMatrixT<T>::addSquare(BaseMatrixT& b, T p) {
  applyBinary(binary::AddSquare<T>(p), b);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(DecayAddSquare, TWO_PARAMETER,
                                  a = p1 * a + p2 * b * b);
template<class T>
void BaseMatrixT<T>::decayAddSquare(BaseMatrixT& b, T p1, T p2) {
  applyBinary(binary::DecayAddSquare<T>(p1, p2), b);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(DecayAddSquareMul, TWO_PARAMETER,
                                   a = p1 * a + p2 * b * b * c * c);
template<class T>
void BaseMatrixT<T>::decayAddSquareMul(BaseMatrixT& b, BaseMatrixT& c, T p1,
                                       T p2) {
  applyTernary(ternary::DecayAddSquareMul<T>(p1, p2), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(ReciprocalSum, THREE_PARAMETER,
                                   a = 1 / (p1 * b + p2 * c + p3));
template<class T>
void BaseMatrixT<T>::reciprocalSum(BaseMatrixT& b, BaseMatrixT& c, T p1, T p2,
                                   T p3) {
  applyTernary(ternary::ReciprocalSum<T>(p1, p2, p3), b, c);
}

DEFINE_MATRIX_BINARY_PARAMETER_OP(Reciprocal2, TWO_PARAMETER,
                                  a = 1 / (p1 * b + p2));
template<class T>
void BaseMatrixT<T>::reciprocal2(BaseMatrixT& b, T p1, T p2) {
  applyBinary(binary::Reciprocal2<T>(p1, p2), b);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(DotMulSquareSum, TWO_PARAMETER,
                                   T tmp = p1 * b + p2 * c;
                                   a *= tmp * tmp);
template<class T>
void BaseMatrixT<T>::dotMulSquareSum(BaseMatrixT& b, BaseMatrixT& c, T p1,
                                     T p2) {
  applyTernary(ternary::DotMulSquareSum<T>(p1, p2), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(DotSquareSum, TWO_PARAMETER,
                                   T tmp = p1 * b + p2 * c;
                                   a = tmp * tmp);
template<class T>
void BaseMatrixT<T>::dotSquareSum(BaseMatrixT& b, BaseMatrixT& c, T p1, T p2) {
  applyTernary(ternary::DotSquareSum<T>(p1, p2), b, c);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(DotMulSum, TWO_PARAMETER,
                                   a *= p1 * b + p2 * c);
template<class T>
void BaseMatrixT<T>::dotMulSum(BaseMatrixT& b, BaseMatrixT& c, T p1, T p2) {
  applyTernary(ternary::DotMulSum<T>(p1, p2), b, c);
}

DEFINE_MATRIX_BINARY_OP(CopyAndClear, b = a; a = 0);
template<class T>
void BaseMatrixT<T>::copyAndClear(BaseMatrixT& b) {
  applyBinary(binary::CopyAndClear<T>(), b);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(AddDotMul, TWO_PARAMETER,
                                   a = p1 * a + p2 * b * c);
template<class T>
void BaseMatrixT<T>::addDotMul(BaseMatrixT& b, BaseMatrixT& c, T p1, T p2) {
  applyTernary(ternary::AddDotMul<T>(p1, p2), b, c);
}

DEFINE_MATRIX_BINARY_OP(Assign, a = b;);
template<class T>
void BaseMatrixT<T>::assign(BaseMatrixT& b) {
  if (useGpu_) {
    applyBinary(binary::Assign<T>(), b);
  } else {  // cpu version
    CHECK_EQ(this->height_, b.height_);
    CHECK_EQ(this->width_, b.width_);
    memcpy(data_, b.data_, sizeof(T) * height_ * width_);
  }
}

template<class T>
void BaseMatrixT<T>::assignAtOffset(BaseMatrixT& b, int64_t columnOffset) {
  if (columnOffset + b.width_ <= width_) {
    int numRows = height_;
    int numCols = b.width_;
    MatrixOffset offset(columnOffset, 0, 0, 0);
    applyBinary(binary::Assign<T>(), b, numRows, numCols, offset);
  } else if (columnOffset + width_ <= b.width_) {
    int numRows = height_;
    int numCols = width_;
    MatrixOffset offset(0, 0, columnOffset, 0);
    applyBinary(binary::Assign<T>(), b, numRows, numCols, offset);
  } else {
    LOG(FATAL) << "Wrong argument "
               << " a.width=" << width_ << " b.width=" << b.width_
               << " columnOffset=" << columnOffset;
  }
}

DEFINE_MATRIX_BINARY_OP(DeepSwap, T tmp = a; a = b; b = tmp);
template<class T>
void BaseMatrixT<T>::deepSwap(BaseMatrixT& b) {
    applyBinary(binary::DeepSwap<T>(), b);
}

template<>
void BaseMatrixT<real>::rowDotMul(size_t destCol,
                                  BaseMatrixT& b,
                                  BaseMatrixT& c) {
  int numRows = b.height_;
  int numCols = b.width_;
  MatrixOffset offset(destCol, 0, 0, 0, 0, 0);
  aggregate(aggregate::sum(), base::binary::mul(), base::binary::add(), b, c,
            numRows, numCols, offset, false_type(),
            true_type() /*aAsColVector*/);
}

template<class T>
void BaseMatrixT<T>::rowDotMul2(size_t destCol,
                                BaseMatrixT& b,
                                BaseMatrixT& c) {
  CHECK(!useGpu_) << "do not support gpu";

  size_t height = this->height_;
  CHECK_LT(destCol, this->width_);
  CHECK_EQ(height, b.height_);
  CHECK_EQ(height, c.height_);
  CHECK_EQ(b.width_, c.width_);
  size_t width = b.width_;
  T* A = this->data_;
  const T* B = b.data_;
  const T* C = c.data_;
  for (size_t i = 0; i < height;
       ++i, A += this->width_, B += width, C += width) {
    for (size_t j = 0; j < width; ++j) {
      A[destCol] += B[j] * C[j];
    }
  }
}

template<>
void BaseMatrixT<real>::addDotMulVMM(BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  int numRows = b.height_;
  int numCols = b.width_;
  aggregate(aggregate::sum(), base::binary::mul(), base::binary::add(), b, c,
            numRows, numCols, offset, true_type() /*aAsRowVector*/,
            false_type());
}

template<class T>
void BaseMatrixT<T>::addDotMulVMM2(BaseMatrixT& b, BaseMatrixT& c) {
  CHECK(!useGpu_) << "do not support gpu";

  CHECK_EQ(height_, 1LU);
  CHECK_EQ(b.height_, c.height_);
  CHECK_EQ(width_, b.width_);
  CHECK_EQ(width_, c.width_);
  size_t height = b.height_;
  size_t width = b.width_;
  T* A = this->data_;
  const T* B = b.data_;
  const T* C = c.data_;
  for (size_t i = 0; i < height; ++i, B += width, C += width) {
    for (size_t j = 0; j < width; ++j) {
      A[j] += B[j] * C[j];
    }
  }
}

DEFINE_MATRIX_TERNARY_OP(addDotMulMMV, a += b * c);
template<class T>
void BaseMatrixT<T>::addDotMulMMV(BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  int numRows = height_;
  int numCols = width_;
  applyTernary(ternary::addDotMulMMV<T>(), b, c, numRows, numCols, offset,
               true_type() /*cAsRowVector*/, false_type());
}

template<class T>
void BaseMatrixT<T>::addDotMulMMV2(BaseMatrixT& b, BaseMatrixT& c) {
  CHECK(!useGpu_) << "do not support gpu";

  CHECK_EQ(c.height_, 1LU);
  CHECK_EQ(height_, b.height_);
  CHECK_EQ(width_, b.width_);
  CHECK_EQ(width_, c.width_);
  size_t height = height_;
  size_t width = width_;
  T* A = this->data_;
  const T* B = b.data_;
  const T* C = c.data_;
  for (size_t i = 0; i < height; ++i, A += width, B += width) {
    for (size_t j = 0; j < width; ++j) {
      A[j] += B[j] * C[j];
    }
  }
}

template<class T>
void BaseMatrixT<T>::rowScale(size_t cCol, BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, cCol, 0);
  int numRows = height_;
  int numCols = width_;
  applyTernary(ternary::DotMul<T>(), b, c, numRows, numCols, offset,
    false_type(), true_type() /*cAsColVector*/);
}

template<class T>
void BaseMatrixT<T>::rowScale2(size_t cCol, BaseMatrixT& b, BaseMatrixT& c) {
  CHECK(!useGpu_) << "do not support gpu";

  size_t height = this->height_;
  size_t width = this->width_;
  CHECK_EQ(height, b.height_);
  CHECK_EQ(width, b.width_);
  CHECK_LT(cCol, c.width_);
  CHECK_EQ(height, c.height_);
  T* A = this->data_;
  const T* B = b.data_;
  const T* C = c.data_;
  for (size_t i = 0; i < height; ++i, A += width, B += width, C += c.width_) {
    for (size_t j = 0; j < width; ++j) {
      A[j] = B[j] * C[cCol];
    }
  }
}

template<class T>
void BaseMatrixT<T>::colScale(size_t cRow, BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, 0, cRow);
  int numRows = height_;
  int numCols = width_;
  applyTernary(ternary::DotMul<T>(), b, c, numRows, numCols, offset,
               true_type() /* cAsRowVector */, false_type() /* cAsColVector */);
}

template<class T>
void BaseMatrixT<T>::addColScale(size_t cRow, BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, 0, cRow);
  int numRows = height_;
  int numCols = width_;
  applyTernary(ternary::addDotMulMMV<T>(), b, c, numRows, numCols, offset,
               true_type() /* cAsRowVector */, false_type() /* cAsColVector */);
}

template<class T>
void BaseMatrixT<T>::addRowScale(size_t cCol, BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, cCol, 0);
  int numRows = height_;
  int numCols = width_;
  applyTernary(ternary::addDotMulMMV<T>(), b, c, numRows, numCols, offset,
               false_type(), true_type() /*cAsColVector*/);
}

DEFINE_MATRIX_TERNARY_PARAMETER_OP(RowAdd, ONE_PARAMETER, a = b + p * c);
template<class T>
void BaseMatrixT<T>::rowAdd(size_t cCol, BaseMatrixT& b, BaseMatrixT& c, T p) {
  MatrixOffset offset(0, 0, 0, 0, cCol, 0);
  int numRows = height_;
  int numCols = width_;
  applyTernary(ternary::RowAdd<T>(p), b, c, numRows, numCols, offset,
    false_type(), true_type() /*cAsColVector*/);
}

DEFINE_MATRIX_TERNARY_OP(RowPow, a = pow(b, c));
template<>
void BaseMatrixT<real>::rowPow(size_t cCol, BaseMatrixT& b, BaseMatrixT& c) {
  if (useGpu_) {
    MatrixOffset offset(0, 0, 0, 0, cCol, 0);
    int numRows = height_;
    int numCols = width_;
    applyTernary(ternary::RowPow<real>(), b, c, numRows, numCols, offset,
                 false_type(), true_type() /*cAsColVector*/);
  } else {
    size_t height = this->height_;
    size_t width = this->width_;
    CHECK_EQ(height, b.height_);
    CHECK_EQ(width, b.width_);
    CHECK_LT(cCol, c.width_);
    CHECK_EQ(height, c.height_);
    real* A = this->data_;
    const real* B = b.data_;
    const real* C = c.data_;
    for (size_t i = 0; i < height; ++i, A += width, B += width, C += c.width_) {
      vPow(width, B, C[cCol], A);
    }
  }
}

template<class T>
void BaseMatrixT<T>::mulRowVector(BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0);
  int numRows = height_;
  int numCols = width_;
  applyBinary(binary::DotMul<T>(), b, numRows, numCols, offset,
              true_type() /* bAsRowVector */, false_type());
}

DEFINE_MATRIX_BINARY_OP(DotDiv, a /= b);
template<class T>
void BaseMatrixT<T>::divRowVector(BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0);
  int numRows = height_;
  int numCols = width_;
  applyBinary(binary::DotDiv<T>(), b, numRows, numCols, offset,
              true_type() /* bAsRowVector */, false_type());
}

template<>
template <class Agg>
int BaseMatrixT<real>::applyRow(Agg agg, BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  size_t numRows = b.height_;
  size_t numCols = b.width_;
  CHECK_EQ(height_, numRows);
  CHECK_EQ(width_, 1UL);
  aggregate(agg, base::unary::identity(), base::binary::second(), b, numRows,
            numCols, offset, false_type(), true_type() /*aAsColVector*/);

  return 0;
}

template<>
template <class Agg, class Saver>
int BaseMatrixT<real>::applyRow(Agg agg, Saver sv, BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  size_t numRows = b.height_;
  size_t numCols = b.width_;
  CHECK_EQ(height_, numRows);
  CHECK_EQ(width_, 1UL);
  aggregate(agg, base::unary::identity(), sv, b, numRows, numCols, offset,
            false_type(), true_type() /*aAsColVector*/);

  return 0;
}

template<>
template <class Agg>
int BaseMatrixT<real>::applyRow(
     Agg agg, real scaleDest, real scaleAgg, BaseMatrixT& b) {
  if (scaleDest != 0) {
    applyRow(agg, base::binary::add2(scaleDest, scaleAgg), b);
  } else {
    applyRow(agg, base::binary::second(), b);
    if (scaleAgg != 1) {
      mulScalar(scaleAgg);
    }
  }
  return 0;
}

template<>
template <class Agg, class Op, class Saver>
int BaseMatrixT<real>::applyRow(Agg agg, Op op, Saver sv,
                                BaseMatrixT& b, BaseMatrixT& c) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  size_t numRows = b.height_;
  size_t numCols = b.width_;
  CHECK_EQ(height_, numRows);
  CHECK_EQ(width_, 1UL);
  CHECK_EQ(c.height_, numRows);
  CHECK_EQ(c.width_, numCols);
  aggregate(agg, op, sv,
            b, c, numRows, numCols, offset,
            false_type(), true_type() /*aAsColVector*/);
  return 0;
}

template<>
template <class Agg, class Op>
int BaseMatrixT<real>::applyRow(Agg agg, Op op, real scaleDest, real scaleAgg,
                                BaseMatrixT& b, BaseMatrixT& c) {
  if (scaleDest != 0) {
    applyRow(agg, op, base::binary::add2(scaleDest, scaleAgg), b, c);
  } else {
    applyRow(agg, op, base::binary::second(), b, c);
    if (scaleAgg != 1) {
      mulScalar(scaleAgg);
    }
  }
  return 0;
}

template<>
template <class Agg>
int BaseMatrixT<real>::applyCol(Agg agg, BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  size_t numRows = b.height_;
  size_t numCols = b.width_;
  CHECK_EQ(width_, numCols);
  CHECK_EQ(height_, 1UL);
  aggregate(agg, base::unary::identity(), base::binary::second(), b, numRows,
            numCols, offset, true_type() /*aAsRowVector*/, false_type());

  return 0;
}

template<>
template <class Agg, class Saver>
int BaseMatrixT<real>::applyCol(Agg agg, Saver sv, BaseMatrixT& b) {
  MatrixOffset offset(0, 0, 0, 0, 0, 0);
  size_t numRows = b.height_;
  size_t numCols = b.width_;
  CHECK_EQ(width_, numCols);
  CHECK_EQ(height_, 1UL);
  aggregate(agg, base::unary::identity(), sv, b, numRows, numCols, offset,
            true_type() /*aAsRowVector*/, false_type());

  return 0;
}

template<>
template <class Agg>
int BaseMatrixT<real>::applyCol(
     Agg agg, real scaleDest, real scaleAgg, BaseMatrixT& b) {
  if (scaleDest != 0) {
    applyCol(agg, base::binary::add2(scaleDest, scaleAgg), b);
  } else {
    applyCol(agg, base::binary::second(), b);
    if (scaleAgg != 1) {
      mulScalar(scaleAgg);
    }
  }
  return 0;
}

template<>
void BaseMatrixT<real>::sumRows(BaseMatrixT& b, real scaleSum, real scaleDest) {
  applyRow(aggregate::sum(), scaleDest, scaleSum, b);
}

template<>
void BaseMatrixT<real>::maxRows(BaseMatrixT& b) {
  applyRow(aggregate::max(), b);
}

template<>
void BaseMatrixT<real>::minRows(BaseMatrixT& b) {
  applyRow(aggregate::min(), b);
}

template<>
void BaseMatrixT<real>::maxCols(BaseMatrixT& b) {
  applyCol(aggregate::max(), b);
}

template<>
void BaseMatrixT<real>::minCols(BaseMatrixT& b) {
  applyCol(aggregate::min(), b);
}

template<>
void BaseMatrixT<real>::sumCols(BaseMatrixT& b, real scaleSum, real scaleDest) {
  applyCol(aggregate::sum(), scaleDest, scaleSum, b);
}

template<>
void BaseMatrixT<real>::sumOfSquaredDiffs(
    BaseMatrixT& b, BaseMatrixT& c, real scaleSum, real scaleDest) {
  applyRow(aggregate::sum(), base::binary::squaredDiff(),
           scaleDest, scaleSum, b, c);
}

template<>
void BaseMatrixT<real>::sumOfProducts(
    BaseMatrixT& b, BaseMatrixT& c, real scaleSum, real scaleDest) {
  applyRow(aggregate::sum(), base::binary::mul(),
           scaleDest, scaleSum, b, c);
}

template class BaseMatrixT<real>;
template class BaseMatrixT<int>;
}  // namespace paddle
