#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <gtest/gtest.h>
#include "paddle/math/Matrix.h"
#include "TensorCheck.h"

using paddle::Matrix;
using paddle::CpuMatrix;
using paddle::GpuMatrix;
using paddle::CpuVector;
using paddle::GpuVector;
using paddle::CpuIVector;
using paddle::GpuIVector;
using autotest::TensorCheckEqual;
using autotest::TensorCheckErr;

#define INIT_UNARY(A1, A2)                  \
    Tensor A1(height, width);               \
    Tensor A2(height, width);               \
    A1.randomizeUniform();                  \
    A2.copyFrom(A1)
#define INIT_BINARY(A1, A2, B)              \
    INIT_UNARY(A1, A2);                     \
    Tensor B(height, width);                \
    B.randomizeUniform()
#define INIT_TERNARY(A1, A2, B, C)          \
    INIT_BINARY(A1, A2, B);                 \
    Tensor C(height, width);                \
    C.randomizeUniform()
#define INIT_QUATERNARY(A1, A2, B, C, D)    \
    INIT_TERNARY(A1, A2, B, C);             \
    Tensor D(height, width);                \
    D.randomizeUniform()

template<typename Tensor>
struct TestUnaryMatrix {
  typedef std::function<void(Tensor& A1, Tensor& A2)> UnaryFunc;

  explicit TestUnaryMatrix(UnaryFunc testUnaryFunc) {
    for (auto height : {1, 11, 73, 128, 200, 330}) {
      for (auto width : {1, 32, 100, 512, 1000, 3210}) {
        LOG(INFO) << " height=" << height << " width=" << width;
        INIT_UNARY(A1, A2);
        testUnaryFunc(A1, A2);
      }
    }
  }
};

template<typename Tensor>
struct TestBinaryMatrix {
  typedef std::function<void(Tensor& A1, Tensor& A2, Tensor& B)> BinaryFunc;

  explicit TestBinaryMatrix(BinaryFunc testBinaryFunc) {
    for (auto height : {1, 11, 73, 128, 200, 330}) {
      for (auto width : {1, 32, 100, 512, 1000, 3210}) {
        LOG(INFO) << " height=" << height << " width=" << width;
        INIT_BINARY(A1, A2, B);
        testBinaryFunc(A1, A2, B);
      }
    }
  }
};

template<typename Tensor>
struct TestTernaryMatrix {
  typedef std::function<void(
    Tensor& A1, Tensor& A2, Tensor& B, Tensor& C)> TernaryFunc;

  explicit TestTernaryMatrix(TernaryFunc testTernaryFunc) {
    for (auto height : {1, 11, 73, 128, 200, 330}) {
      for (auto width : {1, 32, 100, 512, 1000, 3210}) {
        LOG(INFO) << " height=" << height << " width=" << width;
        INIT_TERNARY(A1, A2, B, C);
        testTernaryFunc(A1, A2, B, C);
      }
    }
  }
};

template<typename Tensor>
struct TestQuaternaryMatrix {
  typedef std::function<void(
    Tensor& A1, Tensor& A2, Tensor& B, Tensor& C, Tensor& D)> QuaternaryFunc;

  explicit TestQuaternaryMatrix(QuaternaryFunc testQuaternaryFunc) {
    for (auto height : {1, 11, 73, 128, 200, 330}) {
      for (auto width : {1, 32, 100, 512, 1000, 3210}) {
        LOG(INFO) << " height=" << height << " width=" << width;
        INIT_QUATERNARY(A1, A2, B, C, D);
        testQuaternaryFunc(A1, A2, B, C, D);
      }
    }
  }
};

template<typename Tensor, class T>
struct TestUnaryVectorT {
  typedef std::function<void(Tensor& A1, Tensor& A2)> UnaryFunc;

  explicit TestUnaryVectorT(UnaryFunc testUnaryFunc) {
    for (auto size : {1, 11, 73, 128, 200, 330, 512, 1000, 4210}) {
      LOG(INFO) << " size=" << size;
      Tensor A1(size);
      Tensor A2(size);
      if (typeid(T) == typeid(real)) {
        A1.rand();
      } else {
        A1.rand(1000);
      }
      A2.copyFrom(A1);
      testUnaryFunc(A1, A2);
    }
  }
};

void SetTensorValue(Matrix& matrix, real value) {
  int height = matrix.getHeight();
  int width = matrix.getWidth();
  int stride = matrix.getStride();
  real* data = matrix.getData();
  for (int i = 0; i < height; i++) {
    int j = rand() % width;  // NOLINT
    if (typeid(matrix) == typeid(CpuMatrix)) {
      data[i * stride + j] = value;
    } else if (typeid(matrix) == typeid(GpuMatrix)) {
      hl_memcpy(&data[i * stride + j], &value, sizeof(real));
    } else {
    }
  }
}

template<typename Tensor>
void testTensorAddScalar(Tensor& A1, Tensor& A2) {
  real p1 = 2.5;
  real p2 = 3.0;
  A1.add(p1);   // a += p
  A2 += p1;
  TensorCheckEqual(A1, A2);

  A1.add(p1, p2);  // a = a * p1 + p2
  A2 = A2 * p1 + p2;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSubScalar(Tensor& A1, Tensor& A2) {
  real p = 2.5;
  A1.subScalar(p);  // a -= p
  A2 -= p;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorMulScalar(Tensor& A1, Tensor& A2) {
  real p = 2.5;
  A1.mulScalar(p);  // a *= p
  A2 *= p;
  TensorCheckEqual(A1, A2);

  real learningRate = 0.7f;
  real decayRate = 1.2f;
  A1.applyL2(learningRate, decayRate);
  A2 = A2 * (1.0f / (1.0f + learningRate * decayRate));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorDivScalar(Tensor& A1, Tensor& A2) {
  real p = 2.5;
  A1.divScalar(p);  // a /= p
  A2 /= p;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorNeg(Tensor& A1, Tensor& A2) {
  A1.neg();  // a = -a
  A2 = -A2;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorAbs(Tensor& A1, Tensor& A2) {
  A1.abs2();  // a = a > 0 ? a : -a
  A2 = A2.abs();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSquare(Tensor& A1, Tensor& A2) {
  A1.square2();  // a = a * a
  A2 = A2.square();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorReciprocal(Tensor& A1, Tensor& A2) {
  A1.reciprocal2();  // a = 1.0f / a
  A2 = A2.reciprocal();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSign(Tensor& A1, Tensor& A2) {
  A1.sign2();  // a = (a > 0) - (a < 0)
  A2 = A2.sign();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorAssign(Tensor& A1, Tensor& A2) {
  A1.assign(1.5);   // a = p
  A2 = A2.constant(1.5);
  TensorCheckEqual(A1, A2);

  A1.one();  // a = 1
  A2 = A2.constant(1.0);
  TensorCheckEqual(A1, A2);

  A1.zero();  // a = 0
  A2 = A2.constant(0.0);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testUnaryBaseOp(Tensor& A1, Tensor& A2) {
  testTensorAddScalar(A1, A2);
  testTensorSubScalar(A1, A2);
  testTensorMulScalar(A1, A2);
  testTensorDivScalar(A1, A2);
  testTensorNeg(A1, A2);
  testTensorAbs(A1, A2);
  testTensorSquare(A1, A2);
  testTensorReciprocal(A1, A2);
  testTensorSign(A1, A2);
  testTensorAssign(A1, A2);
}

template<typename Tensor>
void testUnaryBaseOpInt(Tensor& A1, Tensor& A2) {
  A1.add(2);   // a += p
  A2 += 2;
  TensorCheckEqual(A1, A2);

  A1.add(3, 2);  // a = a * p1 + p2
  A2 = A2 * 3 + 2;
  TensorCheckEqual(A1, A2);

  testTensorNeg(A1, A2);
  testTensorAbs(A1, A2);
}

TEST(Unary, BaseOp) {
  TestUnaryMatrix<CpuMatrix> testCpuMatrix(testUnaryBaseOp<CpuMatrix>);
  TestUnaryVectorT<CpuVector, real> testCpuVector(testUnaryBaseOp<CpuVector>);
  TestUnaryVectorT<CpuIVector, int>
    testCpuIVector(testUnaryBaseOpInt<CpuIVector>);

#ifndef PADDLE_ONLY_CPU
  TestUnaryMatrix<GpuMatrix> testGpuMatrix(testUnaryBaseOp<GpuMatrix>);
  TestUnaryVectorT<GpuVector, real> testGpuVector(testUnaryBaseOp<GpuVector>);
  TestUnaryVectorT<GpuIVector, int>
    testGpuIVector(testUnaryBaseOpInt<GpuIVector>);
#endif
}

template<typename Tensor>
void testTensorExp(Tensor& A1, Tensor& A2) {
  A1.exp2();  // a = exp(a)
  A2 = A2.exp();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorLog(Tensor& A1, Tensor& A2) {
  A1.log2();  // a = log(a)
  A2 = A2.log();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSqrt(Tensor& A1, Tensor& A2) {
  A1.sqrt2();  // a = sqrt(a)
  A2 = A2.sqrt();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorPow(Tensor& A1, Tensor& A2) {
  A1.pow2(3.2);  // a = pow(a, p)
  A2 = A2.pow(3.2);
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testUnayrMathOp(Tensor& A1, Tensor& A2) {
  testTensorExp(A1, A2);
  testTensorLog(A1, A2);
  testTensorSqrt(A1, A2);
  testTensorPow(A1, A2);
}

TEST(Unary, MathOp) {
  TestUnaryMatrix<CpuMatrix> testCpu(testUnayrMathOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestUnaryMatrix<GpuMatrix> testGpu(testUnayrMathOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorClip(Tensor& A1, Tensor& A2) {
  real p1 = 0.003f;
  real p2 = 0.877f;
  A1.clip(p1, p2);  // a = a < p1 ? p1 : (a > p2 ? p2 : a)
  // A2 = A2.min(0.877f).max(0.003f);
  A2 = (A2 < p1).condition(p1, (A2 > p2).condition(p2, A2));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorBiggerThanScalar(Tensor& A1, Tensor& A2) {
  real p = 0.5f;
  A1.biggerThanScalar(p);  // a = a > p ? 1.0f : 0.0f
  A2 = (A2 > p).condition((real)1.0, (real)0.0);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorapplyL1(Tensor& A1, Tensor& A2) {
  /**
   * T lambda = p;
   * a = (a > lambda) ? (a - lambda)
   *                  : (a < -lambda) ? (a + lambda) : 0
   *
   * p = learningRate * decayRate;
   */
  real learningRate = 0.7f;
  real decayRate = 0.6f;
  A1.applyL1(learningRate, decayRate);
  A2 = (A2 > (learningRate * decayRate)).condition(
    (A2 - (learningRate * decayRate)),
    (A2 < -(learningRate * decayRate)).condition(
      (A2 + (learningRate * decayRate)), (real)0.0));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testUnayrCompareOp(Tensor& A1, Tensor& A2) {
  testTensorClip(A1, A2);
  testTensorBiggerThanScalar(A1, A2);

  A1.randomizeUniform();
  A1.subScalar(0.5f);
  A2.copyFrom(A1);
  testTensorapplyL1(A1, A2);
}

TEST(Unary, CompareOp) {
  TestUnaryMatrix<CpuMatrix> testCpu(testUnayrCompareOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestUnaryMatrix<GpuMatrix> testGpu(testUnayrCompareOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorAdd(Tensor& A1, Tensor& A2, Tensor& B) {
  real p1 = 2.5;
  real p2 = 3.2;
  A1.add(B);  // a += b
  A2 += B;
  TensorCheckEqual(A1, A2);

  A1.add(B, p1);  // a += b * p
  A2 += B * p1;
  TensorCheckEqual(A1, A2);

  A1.add(B, p1, p2);  // a = p1 * a + p2 * b
  A2 = A2 * p1 + B * p2;
  TensorCheckEqual(A1, A2);

  A1.addScalar(B, p1);  // a = b + p
  A2 = B + p1;
  TensorCheckEqual(A1, A2);

  A1.addSquare(B, p1);  // a += p * b * b
  A2 += B.constant(p1) * B * B;
  TensorCheckEqual(A1, A2);

  A1.decayAddSquare(B, p1, p2);  // a = p1 * a + p2 * b * b
  A2 = A2 * p1 + B.constant(p2) * B * B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSub(Tensor& A1, Tensor& A2, Tensor& B) {
  real p = 2.5;
  A1.sub(B);  // a -= b
  A2 -= B;
  TensorCheckEqual(A1, A2);

  A1.sub(B, p);  // a -= b * p
  A2 -= B * p;
  TensorCheckEqual(A1, A2);

  A1.subScalar(B, p);  // a = b - p
  A2 = B - p;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorMul(Tensor& A1, Tensor& A2, Tensor& B) {
  real p = 2.5;
  A1.mulScalar(B, p);  // a = b * p
  A2 = B * p;
  TensorCheckEqual(A1, A2);

  A1.dotMulSquare(B);  // a *= b * b
  A2 *= B * B;
  TensorCheckEqual(A1, A2);

  A1.dotSquareMul(B);  // a = a * a * b
  A2 = A2 * A2 * B;
  TensorCheckEqual(A1, A2);

  A1.dotMul(B);  // a *= b
  A2 *= B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorDiv(Tensor& A1, Tensor& A2, Tensor& B) {
  real p = 2.5;
  A1.divScalar(B, p);  // a = b / p
  A2 = B / p;
  TensorCheckEqual(A1, A2);

  A1.scalarDiv(B, p);  // a = p / b
  A2 = B.constant(p) / B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorAssign(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.assign(B);  // a = b
  A2 = B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSquare(Tensor& A1, Tensor& A2, Tensor& B) {
  B.square2(A1);   // b = a * a
  A2 = B.square();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSquareDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.squareDerivative(B);  // a *= 2.0 * b
  A2 = A2 * (real)2.0 * B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorReciprocal(Tensor& A1, Tensor& A2, Tensor& B) {
  B.reciprocal2(A1);  // b = 1.0f / a
  A2 = B.reciprocal();
  TensorCheckEqual(A1, A2);

  real p1 = 0.58;
  real p2 = 0.32;
  A1.reciprocal2(B, p1, p2);  // a = 1 / (p1 * b + p2)
  A2 = (B * p1 + p2).reciprocal();
  TensorCheckEqual(A1, A2);

  real learningRate = 0.7f;
  real decayRate = 1.2f;
  A1.applyL2(B, learningRate, decayRate);  // a *= (1.0f / (1.0f + p * b))
  A2 *= (B.constant(1.0f) +
    B.constant(learningRate * decayRate) * B).reciprocal();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorReciprocalDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.reciprocalDerivative(B);  // a *= -b * b
  A2 *= (-B) * B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSign(Tensor& A1, Tensor& A2, Tensor& B) {
  B.sign2(A1);  // b = a > 0.0f ? 1.0f : -1.0f
  A2 = B.sign();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorAbs(Tensor& A1, Tensor& A2, Tensor& B) {
  B.abs2(A1);  // b = a > 0.0f ? a : -a
  A2 = B.abs();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testBinaryBaseOp(Tensor& A1, Tensor& A2, Tensor& B) {
  testTensorAdd(A1, A2, B);
  testTensorSub(A1, A2, B);
  testTensorMul(A1, A2, B);
  testTensorDiv(A1, A2, B);
  testTensorSquare(A1, A2, B);
  testTensorSquareDerivative(A1, A2, B);
  testTensorReciprocal(A1, A2, B);
  testTensorReciprocalDerivative(A1, A2, B);
  testTensorAbs(A1, A2, B);
  testTensorSign(A1, A2, B);
  testTensorAssign(A1, A2, B);
}

TEST(Binary, BaseOp) {
  TestBinaryMatrix<CpuMatrix> testCpu(testBinaryBaseOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestBinaryMatrix<GpuMatrix> testGpu(testBinaryBaseOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorExp(Tensor& A1, Tensor& A2, Tensor& B) {
  // a = exp(b)
  A1.exp2(B);
  A2 = B.exp();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorExpDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.expDerivative(B);  // a *= b
  A2 *= B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorLog(Tensor& A1, Tensor& A2, Tensor& B) {
  // a = log(b)
  A1.log2(B);
  A2 = B.log();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSqrt(Tensor& A1, Tensor& A2, Tensor& B) {
  // a = sqrt(b)
  A1.sqrt2(B);
  A2 = B.sqrt();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorInvSqrt(Tensor& A1, Tensor& A2, Tensor& B) {
  // a = 1.0f / sqrt(b)
  A1.invSqrt(B);
  A2 = B.sqrt().reciprocal();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorPow(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.pow2(B, 2.5f);  // a = pow(b, p)
  A2 = B.pow(2.5f);
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSoftrelu(Tensor& A1, Tensor& A2, Tensor& B) {
  /*
   * const T THRESHOLD = 40.0;
   * b = log(1.0 +
   *         exp((a > THRESHOLD) ? THRESHOLD
   *             : ((a < -THRESHOLD) ? (-THRESHOLD) : a)))
   */
  B.softrelu(A1);

  real THRESHOLD = 40.0;
  A2 = (B.constant(1.0f) +
        (B > THRESHOLD).condition(
          THRESHOLD, (B < -THRESHOLD).condition(-THRESHOLD, B)).exp()).log();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSoftreluDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  /*
   * const T THRESHOLD = 40.0;
   * a *= (1.0 - exp(-1.0 * ((b > THRESHOLD)
   *                             ? THRESHOLD
   *                             : ((b < -THRESHOLD) ? (-THRESHOLD) : b)))));
   */
  A1.softreluDerivative(B);
  real THRESHOLD = 40.0;
  A2 = A2 * (B.constant(1.0f) -
             (B.constant(-1.0f) *
              (B > THRESHOLD).condition(
                THRESHOLD, (B < -THRESHOLD).condition(-THRESHOLD, B))).exp());
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSigmoid(Tensor& A1, Tensor& A2, Tensor& B) {
  /*
    const T THRESHOLD_MIN = -40.0;
    const T THRESHOLD_MAX = 13.0;
    T tmp = (a < THRESHOLD_MIN) ? THRESHOLD_MIN
            : ((a > THRESHOLD_MAX) ? THRESHOLD_MAX : a);
    b = 1.0f / (1.0f + exp(-tmp)))
   */
  B.sigmoid(A1);

  const real THRESHOLD_MIN = -40.0;
  const real THRESHOLD_MAX = 13.0;
  auto tmp = (B < THRESHOLD_MIN).condition(
    THRESHOLD_MIN, (B > THRESHOLD_MAX).condition(THRESHOLD_MAX, B));
  A2 = (B.constant(1.0f) + (-tmp).exp()).reciprocal();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSigmoidDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.sigmoidDerivative(B);  // a *= b * (1 - b)
  A2 *= B * (B.constant(1.0f) - B);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorTanh(Tensor& A1, Tensor& A2, Tensor& B) {
  B.tanh(A1);  // b = 2.0 / (1.0 + exp(-2 * a)) - 1.0
  A2 = B.constant(2.0f) / ((B * ((real)-2.0f)).exp() + (real)1.0f) - (real)1.0f;
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorTanhDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.tanhDerivative(B);  // a *= 1 - b * b
  A2 *= B.constant(1.0f) - B * B;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorScaledTanh(Tensor& A1, Tensor& A2, Tensor& B) {
  real p1 = 2.5;
  real p2 = 3.1;
  // b = p1 * (2.0 / (1.0 + exp(-2 * p2 * a)) - 1.0)
  B.scaledTanh(A1, p1, p2);
  A2 = B.constant(p1) *
      (B.constant(2.0f) / ((B.constant(-2.0f) * p2 * B).exp() + (real)1.0)
       - (real)1.0);
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorScaledTanhDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  real p1 = 2.5;
  real p2 = 3.1;
  // a *= (p2 / p1) * (p1 * p1 - b * b));
  A1.scaledTanhDerivative(B, p1, p2);
  A2 = A2 * (B.constant(p2 / p1) * (B.constant(p1 * p1) - B * B));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testBinaryMathOp(Tensor& A1, Tensor& A2, Tensor& B) {
  testTensorTanhDerivative(A1, A2, B);
  testTensorScaledTanhDerivative(A1, A2, B);
  testTensorSigmoidDerivative(A1, A2, B);
  testTensorExpDerivative(A1, A2, B);
  testTensorScaledTanh(A1, A2, B);
  testTensorTanh(A1, A2, B);
  testTensorExp(A1, A2, B);
  testTensorLog(A1, A2, B);
  testTensorSqrt(A1, A2, B);
  testTensorInvSqrt(A1, A2, B);
  testTensorPow(A1, A2, B);

  testTensorSoftrelu(A1, A2, B);
  testTensorSoftreluDerivative(A1, A2, B);
  testTensorSigmoid(A1, A2, B);
}

TEST(Binary, MathOp) {
  TestBinaryMatrix<CpuMatrix> testCpu(testBinaryMathOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestBinaryMatrix<GpuMatrix> testGpu(testBinaryMathOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorRelu(Tensor& A1, Tensor& A2, Tensor& B) {
  B.relu(A1);  // b = a > 0.0f ? a : 0.0f
  A2 = (B > (real)0.0f).condition(B, (real)0.0f);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorReluDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.reluDerivative(B);  // a *= (b > 0.0f ? 1.0f : 0.0f)
  A2 *= (B > (real)0.0).condition((real)1.0, (real)0.0);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorBrelu(Tensor& A1, Tensor& A2, Tensor& B) {
  /*
   * b = a > p1 ? a : p1
   * b = b < p2 ? b : p2
   * int p1 = 0, p2 = 24;
   */
  SetTensorValue(B, 32.0f);
  B.brelu(A1);
  auto tmp = (B > (real)0.0f).condition(B, (real)0.0f);
  A2 = (tmp < (real)24.0f).condition(tmp, (real)24.0f);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorBreluDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  SetTensorValue(B, 32.0f);
  /*
   * a *= (b > p1 && b < p2) ? 1.0 : 0.0
   * int p1 = 0, p2 = 24;
   */
  A1.breluDerivative(B);
  A2 *= (B > (real)0.0f && B < (real)24.0f).condition((real)1.0f, (real)0.0f);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorAbsDerivative(Tensor& A1, Tensor& A2, Tensor& B) {
  A1.absDerivative(B);  // a = (b > 0) ? a : (b < 0) ? -a : 0
  A2 = (B > (real)0.0f).condition(A2,
    (B < (real)0.0f).condition(-A2, (real)0.0f));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorIsEqualTo(Tensor& A1, Tensor& A2, Tensor& B) {
  real p = 0.613;
  SetTensorValue(B, p);
  A1.isEqualTo(B, p);  // a = (b == p)
  A2 = (B == p);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorapplyL1(Tensor& A1, Tensor& A2, Tensor& B) {
  /**
   * T lambda = p * b;
   * a = (a > lambda) ? (a - lambda)
   *                  : (a < -lambda) ? (a + lambda) : 0
   *
   * p = learningRate * decayRate;
   */
  real learningRate = 0.7f;
  real decayRate = 0.6f;
  A1.applyL1(B, learningRate, decayRate);
  auto lambda = B.constant(learningRate * decayRate) * B;
  A2 = (A2 > lambda).condition(
    (A2 - lambda), (A2 < -lambda).condition((A2 + lambda), (real)0.0f));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testBinaryCompareOp(Tensor& A1, Tensor& A2, Tensor& B) {
  B.subScalar(0.5f);
  SetTensorValue(B, 0.0f);
  testTensorReluDerivative(A1, A2, B);

  A1.randomizeUniform();
  A2.copyFrom(A1);
  testTensorBreluDerivative(A1, A2, B);

  testTensorAbsDerivative(A1, A2, B);
  testTensorRelu(A1, A2, B);
  testTensorBrelu(A1, A2, B);
  testTensorIsEqualTo(A1, A2, B);
}

TEST(Binary, CompareOp) {
  TestBinaryMatrix<CpuMatrix> testCpu(testBinaryCompareOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestBinaryMatrix<GpuMatrix> testGpu(testBinaryCompareOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorAdd(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.add(B, C);  // a = b + c
  A2 = B + C;
  TensorCheckEqual(A1, A2);

  real p1 = 1.5;
  real p2 = 2.5;
  real p3 = 3.8;
  A1.add(B, p1, C, p2);  // a = p1 * b + p2 * c
  A2 = B * p1 + C * p2;
  TensorCheckEqual(A1, A2);

  A1.add2(B, C);  // a = a + b + c
  A2 = A2 + B + C;
  TensorCheckEqual(A1, A2);

  A1.add2(B, C, p1, p2, p3);  // a = p1 * a + p2 * b + p3 * c
  A2 = A2 * p1 + B * p2 + C * p3;
  TensorCheckEqual(A1, A2);

  A1.decayAddSquareMul(B, C, p1, p2);  // a = p1 * a + p2 * b * b * c * c
  A2 = A2 * p1 + B.constant(p2) * B * B * C * C;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSub(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.sub(B, C);  // a = b - c
  A2 = B - C;
  TensorCheckEqual(A1, A2);

  real p1 = 1.5;
  real p2 = 2.5;
  A1.sub(B, p1, C, p2);  // a = p1 * b - p2 * c
  A2 = B * p1 - C * p2;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorMul(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.dotMul(B, C);  // a = b * c
  A2 = B * C;
  TensorCheckEqual(A1, A2);

  A1.dotMulSquare(B, C);  // a = b * c * c
  A2 = B * C * C;
  TensorCheckEqual(A1, A2);

  A1.dotSquareSquare(B, C);  // a = b * b * c * c
  A2 = B * B * C * C;
  TensorCheckEqual(A1, A2);

  real p1 = 1.5;
  real p2 = 2.5;

  /*
   * T tmp = p1 * b + p2 * c;
   * a *= tmp * tmp
   */
  A1.dotMulSquareSum(B, C, p1, p2);
  auto tmp = B * p1 + C * p2;
  A2 *= tmp * tmp;
  TensorCheckEqual(A1, A2);

  /*
   * T tmp = p1 * b + p2 * c;
   * a = tmp * tmp
   */
  A1.dotSquareSum(B, C, p1, p2);
  auto tmp2 = B * p1 + C * p2;
  A2 = tmp2 * tmp2;
  TensorCheckEqual(A1, A2);

  // a *= p1 * b + p2 * c
  A1.dotMulSum(B, C, p1, p2);
  A2 *= B * p1 + C * p2;
  TensorCheckEqual(A1, A2);

  // a = p1 * a + p2 * b * c
  A1.addDotMul(B, C, p1, p2);
  A2 = A2 * p1 + B.constant(p2) * B * C;
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorDiv(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.dotDiv(B, C);  // a = (b == 0.0) ? 0.0 : b / c
  A2 = (B == (real)0.0).condition((real)0.0, B / C);
  TensorCheckEqual(A1, A2);

  real p1 = 1.5;
  real p2 = 2.5;
  A1.dotDiv(B, C, p1, p2);  // a = (b + p1) / (c + p2)
  A2 = (B + p1) / (C + p2);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorReciprocal(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  real p1 = 1.5;
  real p2 = 2.5;
  real p3 = 3.5;
  A1.reciprocalSum(B, C, p1, p2, p3);  // a = 1 / (p1 * b + p2 * c + p3)
  A2 = (B * p1 + C * p2 + p3).reciprocal();
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorSoftCrossEntropy(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.softCrossEntropy(B, C);  // a = -c * log(b) - (1 - c) * log(1 - b)
  A2 = -C * B.log() - (C.constant(1.0f) - C) * (B.constant(1.0f) - B).log();
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorSoftCrossEntropyBp(Tensor& A1,
                                  Tensor& A2,
                                  Tensor& B,
                                  Tensor& C) {
  A1.softCrossEntropyBp(B, C);  // a += (b - c) / (b * (1 - b))
  A2 += (B - C) / (B * (B.constant(1.0f) - B));
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTernaryBaseOp(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  testTensorAdd(A1, A2, B, C);
  testTensorSub(A1, A2, B, C);
  testTensorMul(A1, A2, B, C);
  testTensorDiv(A1, A2, B, C);
  testTensorReciprocal(A1, A2, B, C);
  testTensorSoftCrossEntropyBp(A1, A2, B, C);

  testTensorSoftCrossEntropy(A1, A2, B, C);
}

TEST(Ternary, BaseOp) {
  TestTernaryMatrix<CpuMatrix> testCpu(testTernaryBaseOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestTernaryMatrix<GpuMatrix> testGpu(testTernaryBaseOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorBinaryLabelCrossEntropy(Tensor& A1,
                                       Tensor& A2,
                                       Tensor& B,
                                       Tensor& C) {
  A1.binaryLabelCrossEntropy(B, C);  // a = c > 0.5 ? -log(b) : -log(1.0 - b)
  A2 = (C > (real)0.5).condition(
    -(B.log()), -((B.constant(1.0f) - B).log()));
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorBinaryLabelCrossEntropyBp(Tensor& A1,
                                         Tensor& A2,
                                         Tensor& B,
                                         Tensor& C) {
  // a += c > 0.5 ? -1.0 / b : 1.0 / (1.0 - b)
  A1.binaryLabelCrossEntropyBp(B, C);
  A2 += (C > (real)0.5).condition(
    (B.constant(-1.0f) / B), (B.constant(1.0f) - B).reciprocal());
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorLogisticRegressionLoss(Tensor& A1,
                                      Tensor& A2,
                                      Tensor& B,
                                      Tensor& C) {
  SetTensorValue(B, 50.0f);
  SetTensorValue(B, -50.0f);
  /**
   * const T THRESHOLD = 40.0;
   * T x = (b > THRESHOLD) ? THRESHOLD : (b < -THRESHOLD)
   *                                        ? -THRESHOLD
   *                                        : b;
   * a = log(1 + exp(x)) - c * x
   */
  A1.logisticRegressionLoss(B, C);
  real THRESHOLD = 40.0;
  auto tmp = (B > THRESHOLD).condition(
    THRESHOLD, (B < -THRESHOLD).condition(-THRESHOLD, B));
  A2 = (C.constant(1.0f) + tmp.exp()).log() - C * tmp;
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorLogisticRegressionLossBp(Tensor& A1,
                                        Tensor& A2,
                                        Tensor& B,
                                        Tensor& C) {
  SetTensorValue(B, 50.0f);
  SetTensorValue(B, -50.0f);
  /**
   * const T THRESHOLD = 40.0;
   * T x = (b > THRESHOLD) ? THRESHOLD : (b < -THRESHOLD)
   *                                        ? -THRESHOLD
   *                                        : b;
   * x = exp(x); a = x / (1 + x) - c
   */
  A1.logisticRegressionLossBp(B, C);
  real THRESHOLD = 40.0;
  auto tmp = (B > THRESHOLD).condition(
    THRESHOLD, (B < -THRESHOLD).condition(-THRESHOLD, B));
  auto tmp2 = tmp.exp();
  A2 = tmp2 / (C.constant(1.0) + tmp2) - C;
  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorBiggerThan(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.biggerThan(B, C);  // a = (b > c) ? 1.0f : 0.0f
  A2 = (B > C).condition((real)1.0f, (real)0.0f);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorMax(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  A1.max2(B, C);  // a = (b > c) ? b : c
  A2 = (B > C).condition(B, C);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTernaryCompareOp(Tensor& A1, Tensor& A2, Tensor& B, Tensor& C) {
  testTensorBinaryLabelCrossEntropyBp(A1, A2, B, C);
  testTensorBinaryLabelCrossEntropy(A1, A2, B, C);
  testTensorBiggerThan(A1, A2, B, C);
  testTensorMax(A1, A2, B, C);

  testTensorLogisticRegressionLoss(A1, A2, B, C);
  testTensorLogisticRegressionLossBp(A1, A2, B, C);
}

TEST(Ternary, CompareOp) {
  TestTernaryMatrix<CpuMatrix> testCpu(testTernaryCompareOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestTernaryMatrix<GpuMatrix> testGpu(testTernaryCompareOp<GpuMatrix>);
#endif
}

template<typename Tensor>
void testQuaternaryAdd(Tensor& A1,
                       Tensor& A2,
                       Tensor& B,
                       Tensor& C,
                       Tensor& D) {
  // A1.add3(B, C, D, 1.5f, 2.5f, 3.5f);  // a = p1 * b + p2 * c + p3 * d
  // A2 = B * 1.5f + C * 2.5f + D * 3.5f;
  // TensorCheckEqual(A1, A2);

  /*
   * T tmp = p1 * b + p2 * c + p3 * d;
   * a += tmp * tmp
   */
  real p1 = 1.5f;
  real p2 = 2.5f;
  real p3 = 3.5f;
  A1.addSquareSum(B, C, D, p1, p2, p3);
  auto tmp = B * p1 + C * p2 + D * p3;
  A2 += tmp * tmp;
  TensorCheckEqual(A1, A2);
}

TEST(Quaternary, BaseOp) {
  TestQuaternaryMatrix<CpuMatrix> testCpu(testQuaternaryAdd<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestQuaternaryMatrix<GpuMatrix> testGpu(testQuaternaryAdd<GpuMatrix>);
#endif
}

template<typename Tensor>
void testTensorBiggerThan(Tensor& A1,
                          Tensor& A2,
                          Tensor& B,
                          Tensor& C,
                          Tensor& D) {
  // a = ((b > c && d > 0.5f) || (b < c && d < 0.5f)) ? 1.0f : 0.0f);
  A1.biggerThan(B, C, D);
  A2 = ((B > C && D > (real)0.5)
        || (B < C && D < (real)0.5)).condition((real)1.0, (real)0.0);
  TensorCheckEqual(A1, A2);
}

template<typename Tensor>
void testTensorRankLoss(Tensor& A1,
                        Tensor& A2,
                        Tensor& B,
                        Tensor& C,
                        Tensor& D) {
  /**
   * const T THRESHOLD = 40.0; a = b - c;
   * a = (a > THRESHOLD)
   *         ? THRESHOLD
   *         : ((a < -THRESHOLD) ? (-THRESHOLD) : a);
   * a = log(1 + exp(a)) - a * d
   */
  A1.rankLoss(B, C, D);

  real THRESHOLD = 40.0;
  auto tmp = B - C;
  auto tmp2 = (tmp > THRESHOLD).condition(
    THRESHOLD, (tmp < -THRESHOLD).condition(-THRESHOLD, tmp));
  A2 = (D.constant(1.0f) + tmp2.exp()).log() - tmp2 * D;

  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testTensorRankLossBp(Tensor& A1,
                          Tensor& A2,
                          Tensor& B,
                          Tensor& C,
                          Tensor& D) {
  /**
   * const T THRESHOLD = 40.0; a = b - c;
   * a = (a > THRESHOLD)
   *         ? THRESHOLD
   *         : ((a < -THRESHOLD) ? (-THRESHOLD) : a);
   * a = exp(a); a = (a / (1 + a) - d)
   */
  A1.rankLossBp(B, C, D);
  real THRESHOLD = 40.0;
  auto tmp = B - C;
  auto tmp2 = (tmp > THRESHOLD).condition(
    THRESHOLD, (tmp < -THRESHOLD).condition(-THRESHOLD, tmp));
  auto tmp3 = tmp2.exp();
  A2 = tmp3 / (D.constant(1.0f) + tmp3) - D;

  TensorCheckErr(A1, A2);
}

template<typename Tensor>
void testQuaternaryCompareOp(Tensor& A1,
                             Tensor& A2,
                             Tensor& B,
                             Tensor& C,
                             Tensor& D) {
  testTensorBiggerThan(A1, A2, B, C, D);
  testTensorRankLoss(A1, A2, B, C, D);
  testTensorRankLossBp(A1, A2, B, C, D);
}

TEST(Quaternary, CompareOp) {
  TestQuaternaryMatrix<CpuMatrix> testCpu(testQuaternaryCompareOp<CpuMatrix>);

#ifndef PADDLE_ONLY_CPU
  TestQuaternaryMatrix<GpuMatrix> testGpu(testQuaternaryCompareOp<GpuMatrix>);
#endif
}

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  hl_start();
  hl_init(0);
  return RUN_ALL_TESTS();
}

