#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/utils/Logging.h"
#include "BaseMatrix.h"
#include "TrainingAlgorithmOp.h"

#if __cplusplus > 199711L

#include "TensorAssign.h"

namespace paddle {

void sparseMomentumApply(BaseMatrix& value,
                         BaseMatrix& grad,
                         BaseMatrix& momU,
                         BaseMatrix& momV,
                         real alpha,
                         real beta,
                         real gamma,
                         real tau,
                         real learningRate) {
  auto expr1 = momU.lazyAssign(momU - (alpha * gamma * learningRate) * grad);
  auto expr2 = momV.lazyAssign(
    momV + (tau * alpha * gamma * learningRate) * grad);
  auto expr3 = value.lazyAssign(
    (tau / beta + (real)1 / alpha) * momU + ((real)1 / beta) * momV);

  AssignEvaluate(expr1, expr2, expr3);
}

void adadeltaApply(BaseMatrix& value,
                   BaseMatrix& grad,
                   BaseMatrix& mom,
                   BaseMatrix& accum,
                   BaseMatrix& accum_update,
                   BaseMatrix& lr,
                   real rou,
                   real epsilon,
                   real learningRate,
                   real momentum,
                   real decayRate) {
  auto expr1 = accum.lazyAssign(rou * accum + ((real)1 - rou) * grad.square());
  auto expr2 = lr.lazyAssign(
    ((accum_update + epsilon) / (accum + epsilon)).sqrt());
  auto expr3 = accum_update.lazyAssign(
    rou * accum_update + ((real)1 - rou) * (grad * lr).square());
  auto expr4 = mom.lazyAssign(
    mom * momentum - learningRate * lr * (grad + value * decayRate));
  auto expr5 = value.lazyAssign(value + mom);

  AssignEvaluate(expr1, expr2, expr3, expr4, expr5);
}

void adagradApply(BaseMatrix& value,
                  BaseMatrix& grad,
                  BaseMatrix& mom,
                  BaseMatrix& accum_buffer,
                  BaseMatrix& accum,
                  BaseMatrix& lr,
                  real epsilon,
                  real learningRate,
                  real momentum,
                  real decayRate) {
  auto expr1 = accum.lazyAssign(accum + grad.square());
  auto expr2 = lr.lazyAssign(
    (accum_buffer + accum + epsilon).sqrt().reciprocal());
  auto expr3 = mom.lazyAssign(
    mom * momentum - learningRate * lr * (grad + value * decayRate));
  auto expr4 = value.lazyAssign(value + mom);

  AssignEvaluate(expr1, expr2, expr3, expr4);
}

void rmspropApply(BaseMatrix& value,
                  BaseMatrix& grad,
                  BaseMatrix& mom,
                  BaseMatrix& g,
                  BaseMatrix& f,
                  BaseMatrix& lr,
                  real accumulatedRou,
                  real rou,
                  real epsilon,
                  real learningRate,
                  real momentum,
                  real decayRate,
                  bool firstTime) {
  auto expr2 = f.lazyAssign(accumulatedRou * f + ((real)1 - rou) * grad);
  auto expr3 = lr.lazyAssign((g - f.square() + epsilon).sqrt().reciprocal());
  auto expr4 = mom.lazyAssign(
    mom * momentum - learningRate * lr * (grad + value * decayRate));
  auto expr5 = value.lazyAssign(value + mom);

  if (firstTime) {
    auto expr1 = g.lazyAssign(accumulatedRou * g + grad.square());

    AssignEvaluate(expr1, expr2, expr3, expr4, expr5);
  } else {
    auto expr1 = g.lazyAssign(
      accumulatedRou * g + ((real)1 - rou) * grad.square());

    AssignEvaluate(expr1, expr2, expr3, expr4, expr5);
  }
}

void decayedAdagradApply(BaseMatrix& value,
                         BaseMatrix& grad,
                         BaseMatrix& mom,
                         BaseMatrix& accum,
                         BaseMatrix& lr,
                         real accumulatedRou,
                         real rou,
                         real epsilon,
                         real learningRate,
                         real momentum,
                         real decayRate,
                         bool firstTime) {
  auto expr2 = lr.lazyAssign((accum + epsilon).sqrt().reciprocal());
  auto expr3 = mom.lazyAssign(
    mom * momentum - learningRate * lr * (grad + value * decayRate));
  auto expr4 = value.lazyAssign(value + mom);

  if (firstTime) {
    auto expr1 = accum.lazyAssign(accumulatedRou * accum + grad.square());

    AssignEvaluate(expr1, expr2, expr3, expr4);
  } else {
    auto expr1 = accum.lazyAssign(
      accumulatedRou * accum + ((real)1 - rou) * grad.square());

    AssignEvaluate(expr1, expr2, expr3, expr4);
  }
}

void adamApply(BaseMatrix& value,
               BaseMatrix& grad,
               BaseMatrix& mom,  // firse moment
               BaseMatrix& v,    // second moment
               real beta1,
               real beta2,
               real beta1_power,
               real beta2_power,
               real epsilon,
               real learningRate) {
  real alpha = learningRate *
      std::sqrt((real)1 - beta2_power) / ((real)1 - beta1_power);

  auto expr1 = mom.lazyAssign(beta1 * mom + ((real)1 - beta1) * grad);
  auto expr2 = v.lazyAssign(beta2 * v + ((real)1 - beta2) * grad.square());
  auto expr3 = value.lazyAssign(
    value - (mom * alpha) / (v.sqrt() + epsilon));

  AssignEvaluate(expr1, expr2, expr3);
}

void adamaxApply(BaseMatrix& value,
                 BaseMatrix& grad,
                 BaseMatrix& mom,  // firse moment
                 BaseMatrix& u,    // weighted infinity norm
                 real beta1,
                 real beta2,
                 int64_t step,
                 real alpha) {
  auto expr1 = mom.lazyAssign(beta1 * mom + ((real)1 - beta1) * grad);
  auto expr2 = u.lazyAssign(
    (beta2 * u > grad.abs()).condition(beta2 * u, grad.abs()));
  auto expr3 = value.lazyAssign(
    value - (alpha / ((real)1 - (real)std::pow(beta1, step))) * (mom / u));

  AssignEvaluate(expr1, expr2, expr3);
}

}  // namespace paddle

#else

namespace paddle {

void sparseMomentumApply(BaseMatrix& value,
                         BaseMatrix& grad,
                         BaseMatrix& momU,
                         BaseMatrix& momV,
                         real alpha,
                         real beta,
                         real gamma,
                         real tau,
                         real learningRate) {
  /**
   * \alpha_t = \alpha_{t-1} / k
   * \beta_t = \beta_{t-1} / (1 + \lambda\gamma_t)
   * u_t = u_{t-1} - \alpha_t \gamma_t g_t
   * v_t = v_{t-1} + \tau_{t-1} \alpha_t \gamma_t g_t
   * \tau_t = \tau_{t-1} + \beta_t / \alpha_t
   */
  momU -= (alpha * gamma * learningRate) * grad;
  momV += (tau * alpha * gamma * learningRate) * grad;
  value = (tau / beta + (real)1 / alpha) * momU + ((real)1 / beta) * momV;
}

void adadeltaApply(BaseMatrix& value,
                   BaseMatrix& grad,
                   BaseMatrix& mom,
                   BaseMatrix& accum,
                   BaseMatrix& accum_update,
                   BaseMatrix& lr,
                   real rou,
                   real epsilon,
                   real learningRate,
                   real momentum,
                   real decayRate) {
  // E(g_t^2) = \rou * E(g_{t-1}^2) + (1-\rou) * g^2
  accum = rou * accum + ((real)1 - rou) * grad.square();

  // learn_rate: sqrt(( E(dx_{t-1}^2) + epsilon ) / ( E(g_t^2) + epsilon ))
  lr = ((accum_update + epsilon) / (accum + epsilon)).sqrt();

  // E(dx_t^2) = \rou * E(dx_{t-1}^2) + (1-\rou) * (-g*learn_rate)^2
  accum_update = rou * accum_update + ((real)1 - rou) * (grad * lr).square();

  mom = mom * momentum - learningRate * lr * (grad + value * decayRate);
  value += mom;
}

void adagradApply(BaseMatrix& value,
                  BaseMatrix& grad,
                  BaseMatrix& mom,
                  BaseMatrix& accum_buffer,
                  BaseMatrix& accum,
                  BaseMatrix& lr,
                  real epsilon,
                  real learningRate,
                  real momentum,
                  real decayRate) {
  accum += grad.square();
  lr = (accum_buffer + accum + epsilon).sqrt().reciprocal();
  mom = mom * momentum - learningRate * lr * (grad + value * decayRate);
  value += mom;
}

void rmspropApply(BaseMatrix& value,
                  BaseMatrix& grad,
                  BaseMatrix& mom,
                  BaseMatrix& g,
                  BaseMatrix& f,
                  BaseMatrix& lr,
                  real accumulatedRou,
                  real rou,
                  real epsilon,
                  real learningRate,
                  real momentum,
                  real decayRate,
                  bool firstTime) {
  // E(g_t^2) = \rou * E(g_{t-1}^2) + (1-\rou) * g^2
  // For the first time update, make the sum be the current square
  // so that the initial estimation of E(g_t^2) will not be too small.
  if (firstTime) {
    g = accumulatedRou * g + grad.square();
  } else {
    g = accumulatedRou * g + ((real)1 - rou) * grad.square();
  }

  // E(f_t) = \rou * E(f_{t-1}) + (1-\rou) * g
  f = accumulatedRou * f + ((real)1 - rou) * grad;

  // learn_rate = 1/sqrt( ( E(g_t^2) - (E(f_t))^2 + epsilon )
  // Basiclly if the sign of the gradient changes more often,
  // the learning rate will be decreased.
  lr = (g - f.square() + epsilon).sqrt().reciprocal();

  mom = mom * momentum - learningRate * lr * (grad + value * decayRate);
  value += mom;
}

void decayedAdagradApply(BaseMatrix& value,
                         BaseMatrix& grad,
                         BaseMatrix& mom,
                         BaseMatrix& accum,
                         BaseMatrix& lr,
                         real accumulatedRou,
                         real rou,
                         real epsilon,
                         real learningRate,
                         real momentum,
                         real decayRate,
                         bool firstTime) {
  // E(g_t^2) = \rou * E(g_{t-1}^2) + (1-\rou) * g^2
  // For the first time update, make the sum be the current square
  // so that the initial estimation of E(g_t^2) will not be too small.
  if (firstTime) {
    accum = accumulatedRou * accum + grad.square();
  } else {
    accum = accumulatedRou * accum + ((real)1 - rou) * grad.square();
  }

  // learn_rate = 1/sqrt( ( E(g_t^2) + epsilon )
  // Basiclly if the bigger the magnitude gradient is,
  // the smaller the learning rate will be.
  lr = (accum + epsilon).sqrt().reciprocal();

  mom = mom * momentum - learningRate * lr * (grad + value * decayRate);
  value += mom;
}

void adamApply(BaseMatrix& value,
               BaseMatrix& grad,
               BaseMatrix& mom,  // firse moment
               BaseMatrix& v,    // second moment
               real beta1,
               real beta2,
               real beta1_power,
               real beta2_power,
               real epsilon,
               real learningRate) {
  real alpha = learningRate *
      std::sqrt((real)1 - beta2_power) / ((real)1 - beta1_power);

  // m_t = \beta_1 * m_{t-1} + (1-\beta_1)* g_t;
  mom = beta1 * mom + ((real)1 - beta1) * grad;

  // v_t = \beta_2 * v_{t-1} + (1-\beta_2)* g_{t-1}^2
  v = beta2 * v + ((real)1 - beta2) * grad.square();

  value -=  (mom * alpha) / (v.sqrt() + epsilon);
}

void adamaxApply(BaseMatrix& value,
                 BaseMatrix& grad,
                 BaseMatrix& mom,  // firse moment
                 BaseMatrix& u,    // weighted infinity norm
                 real beta1,
                 real beta2,
                 int64_t step,
                 real alpha) {
  // m_t = \beta_1 * m_{t-1} + (1-\beta_1)* g_t;
  mom = beta1 * mom + ((real)1 - beta1) * grad;

  // u_t = max(\beta_2*u_{t-1}, abs(g_t))
  u = (beta2 * u > grad.abs()).condition(beta2 * u, grad.abs());

  // \theta_t = \theta_{t-1} - (\alpha/(1-\beta_1^t))*m_t/u_t
  value -= (alpha / ((real)1 - (real)std::pow(beta1, step))) * (mom / u);
}

}  // namespace paddle

#endif
