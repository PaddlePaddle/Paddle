#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/phi/kernels/multinomial_kernel.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/common/ddim.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/common/data_type.h"
#include "paddle/phi/common/scalar.h"
#include "paddle/phi/core/kernel_registry.h"
#include "paddle/phi/kernels/arg_min_max_kernel.h"
#include "paddle/phi/kernels/empty_kernel.h"
#include "paddle/phi/kernels/funcs/distribution_helper.h"
#include "paddle/phi/kernels/funcs/eigen/common.h"
#include "paddle/phi/kernels/funcs/for_range.h"
#include "paddle/phi/kernels/funcs/inclusive_scan.h"
#include "paddle/phi/kernels/funcs/multinomial_functor.h"
#include "paddle/phi/kernels/top_k_kernel.h"

namespace phi {

template <typename T, typename MT>
__global__ void NormalizeProbability(MT* norm_probs,
                                     const T* in_data,
                                     MT* sum_rows,
                                     int64_t num_distributions,
                                     int64_t num_categories) {
  int id = threadIdx.x + blockIdx.x * blockDim.x +
           blockIdx.y * gridDim.x * blockDim.x;
  if (id < num_distributions * num_categories) {
    PADDLE_ENFORCE(
        static_cast<MT>(in_data[id]) >= 0.0,
        "The input of multinomial distribution should be >= 0, but got %f.",
        static_cast<MT>(in_data[id]));
    int64_t row_id = id / num_categories;
    PADDLE_ENFORCE(sum_rows[row_id] > 0.0,
                   "The sum of one multinomial distribution probability should "
                   "be > 0, but got %f.",
                   sum_rows[row_id]);
    norm_probs[id] = static_cast<MT>(in_data[id]) / sum_rows[row_id];
  }
}

template <typename T>
__device__ int binarySearchFunctor(T* cumulative_probs_data,
                                   T* norm_probs_data,
                                   int num_categories,
                                   T rng_number) {
  int left = 0;
  int right = num_categories;

  while (right - left > 0) {
    int mid = left + (right - left) / 2;

    T temp_prob = cumulative_probs_data[mid];
    if (temp_prob < rng_number) {
      left = mid + 1;
    } else {
      right = mid;
    }
  }

  if (left == num_categories) {
    left = num_categories - 1;
  }

  while (left >= 1 && norm_probs_data[left] == 0) left--;

  return left;
}

template <typename T>
__global__ void sampleMultinomialWithReplacement(
    const int64_t num_samples,
    int64_t* out_data,
    const int64_t num_distributions,
    const int64_t num_categories,
    T* cumulative_probs_data,
    T* norm_probs_data,
    uint64_t seed,
    uint64_t offset) {
  // use binary search to get the selected category sample id.
  // let cumulative_probs_data[id-1] < rng_number < cumulative_probs_data[id].
  size_t idx = gridDim.x * blockDim.x * blockIdx.y + blockDim.x * blockIdx.x +
               threadIdx.x;

#if defined(__NVCC__)
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);
#else
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, idx, offset, &state);
#endif

  int sample = blockIdx.x * blockDim.x + threadIdx.x;
  for (int dist = blockIdx.y; dist < num_distributions; dist += gridDim.y) {
    if (sample < num_samples) {
#if defined(__NVCC__)
      T rng_number = static_cast<T>(hiprand_uniform4(&state).x);
#else
      T rng_number = static_cast<T>(hiprand_uniform4(&state).x);
#endif
      int selected_category =
          binarySearchFunctor<T>(cumulative_probs_data + dist * num_categories,
                                 norm_probs_data + dist * num_categories,
                                 num_categories,
                                 rng_number);

      out_data[sample + dist * num_samples] = selected_category;
    }
  }
}

template <typename T, typename Context>
void MultinomialKernel(const Context& dev_ctx,
                       const DenseTensor& x,
                       const Scalar& num_samples,
                       bool replacement,
                       DenseTensor* out) {
  using MT = typename phi::dtype::MPTypeTrait<T>::Type;

  auto int_num_samples = num_samples.to<int>();
  auto* in_data = x.data<T>();
  int64_t* out_data = dev_ctx.template Alloc<int64_t>(out);
  auto in_dims = x.dims();
  int64_t dim_size = in_dims.size();
  const int64_t num_categories = in_dims[dim_size - 1];
  const int64_t num_distributions = dim_size > 1 ? in_dims[dim_size - 2] : 1;
  // If replacement is False, it's not a replaceable sample. Every category
  // can be used only once.
  if (!replacement) {
    int64_t in_data_numel = x.numel();
    int64_t out_data_numel = out->numel();

    phi::DenseTensor cpu_tensor;
    phi::Copy<Context>(dev_ctx, x, phi::CPUPlace(), false, &cpu_tensor);
    T* cpu_in_data = cpu_tensor.data<T>();
    for (size_t i = 0; i < num_distributions; ++i) {
      int zero_num = 0;
      for (size_t j = 0; j < num_categories; ++j) {
        T weight = cpu_in_data[i * num_categories + j];
        PADDLE_ENFORCE_GE(
            static_cast<MT>(weight),
            0,
            errors::InvalidArgument(
                "Each element of multinomial'input must >= 0, but got %f.",
                static_cast<MT>(weight)));
        if (weight == static_cast<T>(0)) {
          zero_num++;
        }
      }
      int valid_samples = num_categories - zero_num;
      PADDLE_ENFORCE_LE(
          int_num_samples,
          valid_samples,
          errors::InvalidArgument("When replacement=False, 'num_samples' "
                                  "must less than or equal to the number of "
                                  "positive item of input"));
    }

    // Refer to [gumbel softmax algorithm]
    DenseTensor rand = EmptyLike<T, Context>(dev_ctx, x);
    T* rand_data = rand.data<T>();
    funcs::uniform_distribution<MT> dist;
    funcs::exponential_transform<MT> trans(1.0);
    funcs::distribution_and_transform<T>(dev_ctx, &rand, dist, trans);

    funcs::ForRange<Context> for_range(dev_ctx, x.numel());
    for_range([rand_data, in_data] __device__(size_t idx) {
      rand_data[idx] = in_data[idx] / rand_data[idx];
    });

    if (int_num_samples == 1) {
      ArgMaxKernel<T, Context>(
          dev_ctx, rand, -1, true, false, DataType::INT64, out);
    } else {
      std::vector<int64_t> out_dim_vec =
          common::vectorize<int64_t>(out->dims());
      DenseTensor value = Empty<T, Context>(dev_ctx, IntArray(out_dim_vec));
      TopkKernel<T, Context>(
          dev_ctx, rand, num_samples, -1, true, true, &value, out);
    }
    return;
  }

  // Sum of input may not be 1. To get probability in range [0, 1], calculate
  // sum of each row of input, and then use the sum to normalize the input.
  // sum_row_data: sum of each row
  DenseTensor sum_rows_tensor;
  sum_rows_tensor.Resize({num_distributions});
  auto* sum_rows_data = dev_ctx.template Alloc<MT>(&sum_rows_tensor);
  auto& place = *dev_ctx.eigen_device();

  if (num_distributions == 1) {
    auto eigen_input = EigenVector<T>::Flatten(x);
    auto eigen_sum_rows = EigenVector<MT>::Flatten(sum_rows_tensor);
    eigen_sum_rows.device(place) =
        eigen_input.sum(Eigen::DSizes<int, 1>(1))
            .template cast<MT>()
            .eval()
            .template cast<MT>()
            .reshape(Eigen::DSizes<int, 1>(sum_rows_tensor.dims()[0]))
            .template cast<MT>();
  } else {
    auto eigen_input = EigenMatrix<T>::From(x);
    auto eigen_sum_rows = EigenVector<MT>::Flatten(sum_rows_tensor);
    eigen_sum_rows.device(place) =
        eigen_input.sum(Eigen::DSizes<int, 1>(1)).template cast<MT>();
  }
  // Normalize row of each distribution to get the probability in range [0,
  // 1].
  // norm_probs_data: probability of the distribution
  DenseTensor norm_probs_tensor;
  norm_probs_tensor.Resize({num_distributions, num_categories});
  auto* norm_probs_data = dev_ctx.template Alloc<MT>(&norm_probs_tensor);
  // number of threads in a block is min(num_categories, 512)
  int block_size = num_categories < 512 ? num_categories : 512;
  dim3 block_norm(block_size);
  dim3 grid_norm((num_distributions * num_categories - 1) / block_norm.x + 1);

  NormalizeProbability<T, MT>
      <<<grid_norm, block_norm, 0, dev_ctx.stream()>>>(norm_probs_data,
                                                       in_data,
                                                       sum_rows_data,
                                                       num_distributions,
                                                       num_categories);
  // Get cumulative probability of each distribution. It's the same function
  // of ``cumsum`` op.
  DenseTensor cumulative_probs_tensor;
  cumulative_probs_tensor.Resize({num_distributions, num_categories});
  auto* cumulative_probs_data =
      dev_ctx.template Alloc<MT>(&cumulative_probs_tensor);
  // 'phi::funcs::InclusiveScan' has higher accuracy than
  // 'thrust::inclusive_scan'
  funcs::InclusiveScan<MT, std::plus<MT>>(
      /*in*/ norm_probs_data,
      /*out*/ cumulative_probs_data,
      /*outer_dim*/ static_cast<size_t>(num_distributions),
      /*mid_dim*/ static_cast<size_t>(num_categories),
      /*inner_dim*/ static_cast<size_t>(1),
      /*init*/ static_cast<T>(0),
      std::plus<MT>(),
      /*reverse=*/false,
      dev_ctx);
  // Sample the multinomial distributions.
  dim3 block(128);
  int64_t device_id = dev_ctx.GetPlace().GetDeviceId();
  const auto& prop = phi::backends::gpu::GetDeviceProperties(device_id);
  int grid_y = std::min<int64_t>(num_distributions, prop.maxGridSize[1]);
  dim3 grid((int_num_samples - 1) / block.x + 1, grid_y);

  auto gen_cuda = dev_ctx.GetGenerator();
  size_t curand4_loop_times =
      (num_distributions + 4 * grid_y - 1) / (4 * grid_y);
  // 'increment' shoulde be multiple of 4
  uint64_t increment = curand4_loop_times * 4;
  auto seed_offset = gen_cuda->IncrementOffset(increment);

  sampleMultinomialWithReplacement<MT>
      <<<grid, block, 0, dev_ctx.stream()>>>(int_num_samples,
                                             out_data,
                                             num_distributions,
                                             num_categories,
                                             cumulative_probs_data,
                                             norm_probs_data,
                                             seed_offset.first,
                                             seed_offset.second);
}

}  // namespace phi

PD_REGISTER_KERNEL(multinomial,
                   GPU,
                   ALL_LAYOUT,
                   phi::MultinomialKernel,
                   phi::dtype::float16,
                   phi::dtype::bfloat16,
                   float,
                   double) {
  kernel->OutputAt(0).SetDataType(phi::DataType::INT64);
}
