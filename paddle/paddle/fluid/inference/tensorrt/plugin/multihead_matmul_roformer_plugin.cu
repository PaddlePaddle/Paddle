#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/inference/tensorrt/plugin/multihead_matmul_roformer_plugin.h"
#include <stdio.h>
#include <cassert>
#include <hipcub/hipcub.hpp>  // NOLINT
#include <vector>
#include "glog/logging.h"
#include "paddle/fluid/framework/tensor.h"
#include "paddle/fluid/framework/tensor_util.h"
#include "paddle/fluid/inference/tensorrt/plugin/common/common.cuh"
#include "paddle/fluid/inference/tensorrt/plugin/trt_plugin_utils.h"
#include "paddle/fluid/platform/device_context.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/multihead_matmul_functor.h"

namespace paddle {
namespace inference {
namespace tensorrt {
namespace plugin {

// Dynamic Plugin below.
#if IS_TRT_VERSION_GE(6000)

int MultiheadMatmulRoformerPlugin::initialize() TRT_NOEXCEPT { return 0; }

nvinfer1::DimsExprs MultiheadMatmulRoformerPlugin::getOutputDimensions(
    int output_index,
    const nvinfer1::DimsExprs *inputs,
    int nb_inputs,
    nvinfer1::IExprBuilder &expr_builder) TRT_NOEXCEPT {
  // input[0], (B, S, 3 * N * H, 1, 1)
  // input[1], (B, head_num, seq_len, seq_len)
  // output, (B, seq_len, hidden)
  PADDLE_ENFORCE_EQ(output_index,
                    0,
                    platform::errors::InvalidArgument(
                        "There is only one output of the EmbEltwiseLayernorm, "
                        "so the index should be zero,"
                        "but it's (%d)",
                        output_index));
  PADDLE_ENFORCE_EQ(
      nb_inputs,
      4,
      platform::errors::InvalidArgument(
          "The Input of the EmbEltwiseLayernorm should be 3, but we found "
          "it has (%d) inputs",
          nb_inputs));
  nvinfer1::DimsExprs ret;
  ret.nbDims = 3;
  ret.d[0] = inputs[0].d[0];
  ret.d[1] = inputs[0].d[1];
  ret.d[2] = expr_builder.constant(head_size_ * head_number_);
  return ret;
}

bool MultiheadMatmulRoformerPlugin::supportsFormatCombination(
    int pos,
    const nvinfer1::PluginTensorDesc *in_out,
    int nb_inputs,
    int nb_outputs) TRT_NOEXCEPT {
  PADDLE_ENFORCE_NOT_NULL(
      in_out,
      platform::errors::InvalidArgument(
          "The input of swish plugin shoule not be nullptr."));

  PADDLE_ENFORCE_LT(
      pos,
      nb_inputs + nb_outputs,
      platform::errors::InvalidArgument("The pos(%d) should be less than the "
                                        "num(%d) of the input and the output.",
                                        pos,
                                        nb_inputs + nb_outputs));

  const nvinfer1::PluginTensorDesc &in = in_out[pos];
  if (pos == 0) {
    if (with_fp16_) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
      return (in.type == nvinfer1::DataType::kFLOAT ||
              in.type == nvinfer1::DataType::kHALF) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#else
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
#endif
    } else {
      return (in.type == nvinfer1::DataType::kFLOAT) &&
             (in.format == nvinfer1::TensorFormat::kLINEAR);
    }
  }
  const nvinfer1::PluginTensorDesc &prev = in_out[pos - 1];

  if (pos == 1) {
    return in.type == prev.type && in.format == prev.format;
  }

  // output
  return in.type == prev.type && in.format == prev.format;
}

nvinfer1::DataType MultiheadMatmulRoformerPlugin::getOutputDataType(
    int index,
    const nvinfer1::DataType *input_types,
    int nb_inputs) const TRT_NOEXCEPT {
  PADDLE_ENFORCE_EQ(
      index,
      0,
      platform::errors::InvalidArgument(
          "The EmbEltwiseLayernorm Plugin only has one input, so the "
          "index value should be 0, but get %d.",
          index));
  return input_types[0];
}

template <typename T>
__global__ void apply_scale(T *data, T scale, int n) {
#if CUDA_ARCH_FP16_SUPPORTED(__CUDA_ARCH__)
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    data[tid] = data[tid] * scale;
  }
#endif
}

template <typename T>
__global__ void RotrayKernel(const T *inputact,
                             const T *input1,
                             const T *input2,
                             T *output,
                             const int nElement,
                             const int lastdim) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nElement) return;
  T left_elemul_out = input1[index] * inputact[index];
  int col = index % lastdim;
  int half_lastdim = lastdim / 2;
  const int right_index = index - col + (col + half_lastdim) % lastdim;
  output[index] = left_elemul_out + input2[index] * inputact[right_index];
}

inline int round_up(int seq_len, int multiple = 32) {
  PADDLE_ENFORCE_GT(
      multiple,
      0,
      platform::errors::InvalidArgument(
          "multiple should be a positive number, but it's (%d)", multiple));
  return ((seq_len + multiple - 1) / multiple) * multiple;
}

template <typename T>
__global__ void broadcast(const T *src,
                          T *dst,
                          const int seq_len,
                          const int head_num) {
  int batch_id = blockIdx.x / (head_num * seq_len);
  int dst_offset = blockIdx.x * seq_len;
  if (threadIdx.x < seq_len) {
    dst[threadIdx.x + dst_offset] = src[threadIdx.x + batch_id * seq_len];
  }
}

int MultiheadMatmulRoformerPlugin::enqueue(
    const nvinfer1::PluginTensorDesc *input_desc,
    const nvinfer1::PluginTensorDesc *output_desc,
    const void *const *inputs,
    void *const *outputs,
    void *workspace,
    hipStream_t stream) TRT_NOEXCEPT {
  auto input_dims = input_desc[0].dims;
  int input_num = ProductDim(input_dims);
  // input[0], (B, S, 3 * N * H, 1, 1)
  int batch = input_dims.d[0];
  int seq_len = input_dims.d[1];
  phi::DenseTensor multihead_temp_tensor;
  // masks
  int scratch_size = batch * head_number_ * seq_len * seq_len * 1;

  int device_id;
  hipGetDevice(&device_id);
  multihead_temp_tensor.Resize({scratch_size + input_num});
  // for roformer
  phi::DenseTensor temp_roformer_tensor;
  temp_roformer_tensor.Resize({input_num});

  auto input_type = input_desc[0].type;
  if (input_type == nvinfer1::DataType::kFLOAT) {
    VLOG(1) << "TRT Plugin DataType selected. RoformerQkvToContext-->fp32";
    auto *multihead_temp_data = multihead_temp_tensor.mutable_data<float>(
        platform::CUDAPlace(device_id));
    auto *temp_roformer_data =
        temp_roformer_tensor.mutable_data<float>(  // NOLINT
            platform::CUDAPlace(device_id));
    auto *tmp_roformer_ptr = reinterpret_cast<float *>(temp_roformer_data);
    auto *qkptr = multihead_temp_data;
    auto *tptr = multihead_temp_data + scratch_size;

    const float *input0_data = static_cast<const float *>(inputs[0]);
    // fit to [batch, head_num, length, length] + [batch, 1, 1, length]
    phi::DenseTensor temp_qk_bias_tensor;
    float *qk_bias = const_cast<float *>(static_cast<const float *>(inputs[3]));
    if (ProductDim(input_desc[3].dims) == (batch * seq_len)) {
      temp_qk_bias_tensor.Resize({batch, head_number_, seq_len, seq_len});
      auto *temp_qk_bias = temp_qk_bias_tensor.mutable_data<float>(
          platform::CUDAPlace(device_id));
      int grid = batch * head_number_ * seq_len;
      int block = round_up(seq_len);
      broadcast<<<grid, block, 0, stream>>>(
          static_cast<const float *>(inputs[3]),
          temp_qk_bias,
          seq_len,
          head_number_);
      qk_bias = temp_qk_bias;
    }
    const float *input3_data = static_cast<const float *>(qk_bias);
    // BxSx3xNxH => tptr: 3xBxNxSxH.
    TransposeQKV(
        batch, seq_len, head_size_, head_number_, input0_data, tptr, stream);
    hipMemcpy(tmp_roformer_ptr,  // dst
               tptr,              // src
               input_num * sizeof(float),
               hipMemcpyDeviceToDevice);
    int n_q = seq_len * head_number_ * head_size_ * batch;
    constexpr int threads = 128;
    int blocks = (n_q + threads - 1) / threads;
    const float *input_cos_data = static_cast<const float *>(inputs[1]);
    const float *input_sin_data = static_cast<const float *>(inputs[2]);
    RotrayKernel<<<blocks, threads, 0, stream>>>(tmp_roformer_ptr,
                                                 input_cos_data,
                                                 input_sin_data,
                                                 tptr,
                                                 n_q,
                                                 head_size_);  // q
    RotrayKernel<<<blocks, threads, 0, stream>>>(tmp_roformer_ptr + n_q,
                                                 input_cos_data,
                                                 input_sin_data,
                                                 tptr + n_q,
                                                 n_q,
                                                 head_size_);  // k

    auto *device_ctx = static_cast<phi::GPUContext *>(
        platform::DeviceContextPool::Instance().Get(
            platform::CUDAPlace(device_id)));

    const phi::GPUContext &dev_ctx = *device_ctx;
    phi::funcs::MultiheadGPUComputeFunctor<float> multihead_compute_func;
    multihead_compute_func(dev_ctx,
                           batch,
                           seq_len,
                           head_number_,
                           head_size_,
                           qkptr,
                           input3_data,
                           false,
                           tptr,
                           scale_,
                           static_cast<float>(0.0));

    int grid = batch * head_number_ * seq_len;
    int block = head_size_;
    float *output = static_cast<float *>(outputs[0]);
    transpose<float><<<grid, block, 0, stream>>>(
        tptr, output, batch, seq_len, head_number_, head_size_);

  } else if (input_type == nvinfer1::DataType::kHALF) {
#ifdef TRT_PLUGIN_FP16_AVALIABLE
    VLOG(1) << "TRT Plugin DataType selected. QkvToContext-->fp16";
    auto *multihead_temp_data =
        multihead_temp_tensor.mutable_data<int16_t>(  // NOLINT
            platform::CUDAPlace(device_id));

    auto *temp_roformer_data =
        temp_roformer_tensor.mutable_data<int16_t>(  // NOLINT
            platform::CUDAPlace(device_id));
    half *tmp_roformer_ptr = reinterpret_cast<half *>(temp_roformer_data);
    half *qkptr = reinterpret_cast<half *>(multihead_temp_data);
    half *tptr = qkptr + scratch_size;

    const half *input0_data = static_cast<const half *>(inputs[0]);
    // fit to [batch, head_num, length, length] + [batch, 1, 1, length]
    phi::DenseTensor temp_qk_bias_tensor;
    half *qk_bias = const_cast<half *>(static_cast<const half *>(inputs[3]));
    if (ProductDim(input_desc[3].dims) == (batch * seq_len)) {
      temp_qk_bias_tensor.Resize({batch, head_number_, seq_len, seq_len});
      auto *temp_qk_bias =
          reinterpret_cast<half *>(temp_qk_bias_tensor.mutable_data<int16_t>(
              platform::CUDAPlace(device_id)));
      int grid = batch * head_number_ * seq_len;
      int block = round_up(seq_len);
      broadcast<<<grid, block, 0, stream>>>(
          static_cast<const half *>(inputs[3]),
          temp_qk_bias,
          seq_len,
          head_number_);
      qk_bias = temp_qk_bias;
    }
    const half *input3_data = static_cast<const half *>(qk_bias);
    // BxSx3xNxH => tptr: 3xBxNxSxH.
    TransposeQKV(
        batch, seq_len, head_size_, head_number_, input0_data, tptr, stream);
    hipMemcpy(tmp_roformer_ptr,
               tptr,
               input_num * sizeof(half),
               hipMemcpyDeviceToDevice);

    auto *device_ctx = static_cast<phi::GPUContext *>(
        platform::DeviceContextPool::Instance().Get(
            platform::CUDAPlace(device_id)));

    int n_q = seq_len * head_number_ * head_size_ * batch;
    constexpr int threads = 128;
    int blocks = (n_q + threads - 1) / threads;

    const half *input_cos_data = static_cast<const half *>(inputs[1]);
    const half *input_sin_data = static_cast<const half *>(inputs[2]);
    RotrayKernel<<<blocks, threads, 0, stream>>>(tmp_roformer_ptr,
                                                 input_cos_data,
                                                 input_sin_data,
                                                 tptr,
                                                 n_q,
                                                 head_size_);  // q
    RotrayKernel<<<blocks, threads, 0, stream>>>(tmp_roformer_ptr + n_q,
                                                 input_cos_data,
                                                 input_sin_data,
                                                 tptr + n_q,
                                                 n_q,
                                                 head_size_);  // k

    apply_scale<<<blocks, threads, 0, stream>>>(
        tptr, static_cast<half>(scale_), n_q);

    const phi::GPUContext &dev_ctx = *device_ctx;
    phi::funcs::MultiheadGPUComputeFunctor<half> multihead_compute_func;
    multihead_compute_func(dev_ctx,
                           batch,
                           seq_len,
                           head_number_,
                           head_size_,
                           qkptr,
                           input3_data,
                           false,
                           tptr,
                           half(1.),
                           half(0.0));

    int grid = batch * head_number_ * seq_len;
    int block = head_size_;
    half *output = static_cast<half *>(outputs[0]);
    transpose<half><<<grid, block, 0, stream>>>(
        tptr, output, batch, seq_len, head_number_, head_size_);
#else
    PADDLE_THROW(platform::errors::Fatal(
        "The Ernie(Bert) TensorRT Plugin should be "
        "complied with CUDA version >= 10.0 when running with fp16. "
        "Please recomplie it or try to use fp32 by set "
        "config.SetTRTDynamicShapeInfo(min_input_shape, "
        "max_input_shape, opt_input_shape, true"));
#endif
  } else {
    PADDLE_THROW(platform::errors::Fatal(
        "The QKV TRT Plugin's input type should be float or half."));
  }
  return hipGetLastError() != hipSuccess;
}
#endif

}  // namespace plugin
}  // namespace tensorrt
}  // namespace inference
}  // namespace paddle
