/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/tcmpt/cuda/math.h"

// #include "paddle/tcmpt/eigen/scale.h"
// #include "paddle/tcmpt/eigen/sign.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/fluid/platform/float16.h"
#include "paddle/tcmpt/core/convert_utils.h"
#include "paddle/tcmpt/core/kernel_registry.h"

namespace pt {

/**
 * Util Functors
 */

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n)
      : n_inv(static_cast<T>(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};

/**
 * Kernels
 */

template <typename T>
void Sign(const HIPContext& dev_ctx, const DenseTensor& x, DenseTensor* out) {
  eigen::Sign<HIPContext, T>(dev_ctx, x, out);
}

template <typename T>
void Mean(const HIPContext& dev_ctx, const DenseTensor& x, DenseTensor* out) {
  auto size_prob = x.numel();
  const T* x_data = x.data<T>();
  T* out_data = out->mutable_data<T>();
  auto stream = dev_ctx.stream();

  DivideFunctor<T> transformer(size_prob);
  hipcub::TransformInputIterator<T, DivideFunctor<T>, const T*> trans_x(
      x_data, transformer);
  size_t temp_storage_bytes = 0;

  auto err = hipcub::DeviceReduce::Sum(
      nullptr, temp_storage_bytes, trans_x, out_data, size_prob, stream);
  PADDLE_ENFORCE_CUDA_SUCCESS(err);

  pt::DenseTensor tmp(
      TensorMeta(paddle::framework::make_ddim(
                     {static_cast<int64_t>(temp_storage_bytes)}),
                 pt::TransToPtBackend(dev_ctx.GetPlace()),
                 x.type(),
                 x.layout()),
      TensorStatus());
  auto* temp_storage = tmp.mutable_data<uint8_t>();
  err = hipcub::DeviceReduce::Sum(
      temp_storage, temp_storage_bytes, trans_x, out_data, size_prob, stream);
  PADDLE_ENFORCE_CUDA_SUCCESS(err);
}

template <typename T>
void Scale(const HIPContext& dev_ctx,
           const DenseTensor& x,
           float scale,
           float bias,
           bool bias_after_scale,
           DenseTensor* out) {
  eigen::Scale<HIPContext, T>(dev_ctx, x, scale, bias, bias_after_scale, out);
}

template <typename T>
void ScaleSelectedRows(const HIPContext& dev_ctx,
                       const SelectedRowsTensor& x,
                       float scale,
                       float bias,
                       bool bias_after_scale,
                       SelectedRowsTensor* out) {
  out->set_rows(x.rows());
  out->set_height(x.height());
  Scale<T>(
      dev_ctx, x.value(), scale, bias, bias_after_scale, out->mutable_value());
}

template <typename T>
void ScaleHost(const HIPContext& dev_ctx,
               const DenseTensor& x,
               const DenseTensor& scale,
               float bias,
               bool bias_after_scale,
               DenseTensor* out) {
  if (paddle::platform::is_gpu_place(scale.place())) {
    throw std::runtime_error("scale host place error.");
  }
  eigen::Scale<HIPContext, T>(dev_ctx,
                               x,
                               static_cast<float>(*scale.data<T>()),
                               bias,
                               bias_after_scale,
                               out);
}

template <typename T>
void ScaleSelectedRowsHost(const HIPContext& dev_ctx,
                           const SelectedRowsTensor& x,
                           const DenseTensor& scale,
                           float bias,
                           bool bias_after_scale,
                           SelectedRowsTensor* out) {
  out->set_rows(x.rows());
  out->set_height(x.height());
  Scale<T>(dev_ctx,
           x.value(),
           static_cast<float>(*scale.data<T>()),
           bias,
           bias_after_scale,
           out->mutable_value());
}

}  // namespace pt

// TODO(chenweihang): replace by better impl
PT_REGISTER_MODULE(MathCUDA);

using float16 = paddle::platform::float16;
PT_REGISTER_KERNEL("sign", CUDA, NCHW, pt::Sign, float, double, float16) {}
PT_REGISTER_KERNEL("mean", CUDA, NCHW, pt::Mean, float, double, float16) {}
PT_REGISTER_KERNEL("scale",
                   CUDA,
                   NCHW,
                   pt::Scale,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
PT_REGISTER_KERNEL("scale.sr",
                   CUDA,
                   NCHW,
                   pt::ScaleSelectedRows,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {}
PT_REGISTER_KERNEL("scale.host",
                   CUDA,
                   NCHW,
                   pt::ScaleHost,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {
  kernel->InputAt(1).SetBackend(pt::Backend::kCPU);
}
PT_REGISTER_KERNEL("scale.sr.host",
                   CUDA,
                   NCHW,
                   pt::ScaleSelectedRowsHost,
                   float,
                   double,
                   float16,
                   uint8_t,
                   int8_t,
                   int16_t,
                   int,
                   int64_t) {
  kernel->InputAt(1).SetBackend(pt::Backend::kCPU);
}
