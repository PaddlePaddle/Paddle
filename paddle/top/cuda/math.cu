/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/top/cuda/math.h"

#ifdef __NVCC__
#include "hipcub/hipcub.hpp"
#endif
#ifdef __HIPCC__
#include <hipcub/hipcub.hpp>
namespace cub = hipcub;
#endif

#include "paddle/top/core/convert_utils.h"

namespace pt {

/**
 * Util Functors
 */

template <typename T>
struct DivideFunctor {
  HOSTDEVICE explicit inline DivideFunctor(int n)
      : n_inv(static_cast<T>(1.0 / n)) {}

  HOSTDEVICE inline T operator()(const T& x) const { return x * n_inv; }

 private:
  T n_inv;
};

/**
 * Kernels
 */

template <typename T>
void Mean(const CUDADeviceContext& dev_ctx,
          const DenseTensor& x,
          DenseTensor* out) {
  auto size_prob = x.numel();
  const T* x_data = x.data<T>();
  T* out_data = out->mutable_data<T>();
  auto stream = dev_ctx.stream();

  DivideFunctor<T> transformer(size_prob);
  hipcub::TransformInputIterator<T, DivideFunctor<T>, const T*> trans_x(
      x_data, transformer);
  size_t temp_storage_bytes = 0;

  auto err = hipcub::DeviceReduce::Sum(
      nullptr, temp_storage_bytes, trans_x, out_data, size_prob, stream);
  PADDLE_ENFORCE_CUDA_SUCCESS(err);

  pt::DenseTensor tmp(
      TensorMeta(paddle::framework::make_ddim(
                     {static_cast<int64_t>(temp_storage_bytes)}),
                 pt::TransToPtenBackend(dev_ctx.GetPlace()),
                 x.type(),
                 x.layout()),
      TensorStatus());
  auto* temp_storage = tmp.mutable_data<uint8_t>();
  err = hipcub::DeviceReduce::Sum(
      temp_storage, temp_storage_bytes, trans_x, out_data, size_prob, stream);
  PADDLE_ENFORCE_CUDA_SUCCESS(err);
}

template void Mean<float>(const CUDADeviceContext& dev_ctx,
                          const DenseTensor& x,
                          DenseTensor* out);
template void Mean<double>(const CUDADeviceContext& dev_ctx,
                           const DenseTensor& x,
                           DenseTensor* out);
template void Mean<paddle::platform::float16>(const CUDADeviceContext& dev_ctx,
                                              const DenseTensor& x,
                                              DenseTensor* out);

}  // namespace pt
