#define EIGEN_USE_GPU
#include "paddle/framework/op_registry.h"
#include "paddle/operators/softmax_op.h"

REGISTER_OP_GPU_KERNEL(softmax, ops::SoftmaxKernel<ops::GPUPlace, float>);
REGISTER_OP_GPU_KERNEL(softmax_grad, ops::SoftmaxGradKernel<ops::GPUPlace, float>);
