#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/accuracy_op.h"

namespace paddle {
namespace operators {

__global__ void AccuracyCudaKernel(const int N, const int D, const int* Xdata,
                                   const int* labeldata, float* accuracy) {
  int count = 0;
  __shared__ int total;
  total = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N);
       i += blockDim.x * gridDim.x) {
    for (int j = 0; j < D; ++j) {
      if (Xdata[i * D + j] == labeldata[i]) {
        ++count;
        break;
      }
    }
  }
  atomicAdd(&total, count);
  __syncthreads();
  if (threadIdx.x == 0) {
    *accuracy = static_cast<float>(total) / static_cast<float>(N);
  }
}

template <typename T>
class AccuracyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");
    auto* inference = ctx.Input<Tensor>("Inference");
    auto* label = ctx.Input<Tensor>("Label");
    auto* accuracy = ctx.Output<Tensor>("Accuracy");
    // FIXME(typhoonzero): only support indices currently
    // if add support for output values, how to detect the data type?
    const int* inference_data = inference->data<int>();
    const int* label_data = label->data<int>();
    float* accuracy_data = accuracy->mutable_data<float>(ctx.GetPlace());

    size_t num_samples = inference->dims()[0];
    size_t infer_width = inference->dims()[1];
    hipMemset((void**)&accuracy_data, 0, sizeof(float));

    if (num_samples == 0) {
      return;
    }

    int threads = 512;
    int grids = (num_samples + 4096 - 1) / 4096;
    AccuracyCudaKernel<<<grids, threads>>>(
        num_samples, infer_width, inference_data, label_data, accuracy_data);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_GPU_KERNEL(accuracy,
                       paddle::operators::AccuracyOpCUDAKernel<float>);
