#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include "paddle/operators/accuracy_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;

template <int BlockSize>
__global__ void AccuracyCudaKernel(const int N, const int D, const int* Xdata,
                                   const int* labeldata, float* accuracy) {
  int count = 0;
  __shared__ int total[BlockSize];

  // support only 1 block
  for (int i = threadIdx.x; i < (N); i += BlockSize) {
    for (int j = 0; j < D; ++j) {
      if (Xdata[i * D + j] == labeldata[i]) {
        ++count;
        break;
      }
    }
  }
  total[threadIdx.x] = count;
  __syncthreads();

  // reduce the count with init value 0, and output accuracy.
  int result = thrust::reduce(thrust::device, total, total + BlockSize, 0);
  if (threadIdx.x == 0) {
    *accuracy = static_cast<float>(result) / static_cast<float>(N);
  }
}

template <typename T>
class AccuracyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");
    auto* inference = ctx.Input<Tensor>("Inference");
    auto* label = ctx.Input<Tensor>("Label");
    auto* accuracy = ctx.Output<Tensor>("Accuracy");
    // FIXME(typhoonzero): only support indices currently
    // if add support for output values, how to detect the data type?
    const int* inference_data = inference->data<int>();
    const int* label_data = label->data<int>();
    float* accuracy_data = accuracy->mutable_data<float>(ctx.GetPlace());

    size_t num_samples = inference->dims()[0];
    size_t infer_width = inference->dims()[1];
    hipMemset((void**)&accuracy_data, 0, sizeof(float));

    if (num_samples == 0) {
      return;
    }

    AccuracyCudaKernel<PADDLE_CUDA_NUM_THREADS><<<1, PADDLE_CUDA_NUM_THREADS>>>(
        num_samples, infer_width, inference_data, label_data, accuracy_data);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_GPU_KERNEL(accuracy,
                       paddle::operators::AccuracyOpCUDAKernel<float>);
