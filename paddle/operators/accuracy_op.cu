#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/accuracy_op.h"

namespace paddle {
namespace operators {

__global__ void AccuracySingleKernel(const int N, const int D, const int top_k,
                                     const int* Xdata, const int* labelData,
                                     float* accuracy) {
  int correct = 0;
  for (int row = 0; row < N; row++) {
    const int label = labelData[row];
    for (int col = 0; col < D; col++) {
      const int pred = Xdata[row * D + col];
      if (pred == label) {
        ++correct;
        break;
      }
    }
  }
  *accuracy = static_cast<float>(correct) / static_cast<float>(N);
}

template <typename T>
class AccuracyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");
    auto* inference = ctx.Input<Tensor>("Inference");
    auto* label = ctx.Input<Tensor>("Label");
    auto* accuracy = ctx.Output<Tensor>("Accuracy");
    // FIXME(typhoonzero): only support indices currently
    // if add support for output values, how to detect the data type?
    const int* inference_data = inference->data<int>();
    const int* label_data = label->data<int>();
    float* accuracy_data = accuracy->mutable_data<float>(ctx.GetPlace());

    size_t num_samples = inference->dims()[0];
    size_t infer_width = inference->dims()[1];
    hipMemset((void**)&accuracy_data, 0, sizeof(float));

    if (num_samples == 0) {
      return;
    }

    AccuracySingleKernel<<<1, 1>>>(num_samples, infer_width, 1, inference_data,
                                   label_data, accuracy_data);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_GPU_KERNEL(accuracy,
                       paddle::operators::AccuracyOpCUDAKernel<float>);
