#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU

#include "paddle/operators/softmax_with_cross_entropy_op.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

namespace {
template <typename T>
__global__ void CrossEntropyGrad(T* out_grad, const T* in_grad,
                                 const int* labels, const int batch_size,
                                 const int class_num) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int sample_idx = tid / class_num;

  if (tid < batch_size * class_num) out_grad[tid] *= in_grad[sample_idx];
  __syncthreads();

  if (tid < batch_size) {
    PADDLE_ASSERT(labels[sample_idx] >= 0 && labels[sample_idx] < class_num);
    out_grad[tid * class_num + labels[tid]] -= 1.;
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* logit_grad,
                                               const T* loss_grad,
                                               const T* labels,
                                               const int batch_size,
                                               const int class_num) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < batch_size * class_num) {
    int row_ids = ids / class_num;
    logit_grad[ids] = logit_grad[ids] * loss_grad[row_ids] - labels[ids];
  }
}
}  // namespace

template <typename T>
class SoftmaxWithCrossEntropyCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* logits = context.Input<Tensor>("Logits");
    const Tensor* labels = context.Input<Tensor>("Label");
    Tensor* softmax = context.Output<Tensor>("Softmax");

    Tensor* loss = context.Output<Tensor>("Loss");
    softmax->mutable_data<T>(context.GetPlace());
    loss->mutable_data<T>(context.GetPlace());

    math::SoftmaxFunctor<platform::GPUPlace, T>()(context.device_context(),
                                                  logits, softmax);
    math::CrossEntropyFunctor<platform::GPUPlace, T>()(
        context.device_context(), loss, softmax, labels,
        context.Attr<bool>("softLabel"));
  }
};

template <typename T>
class SoftmaxWithCrossEntropyGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(context.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* labels = context.Input<Tensor>("Label");
    const T* loss_grad_data =
        context.Input<Tensor>(framework::GradVarName("Loss"))->data<T>();
    Tensor* logit_grad =
        context.Output<Tensor>(framework::GradVarName("Logits"));
    logit_grad->ShareDataWith<T>(*context.Input<Tensor>("Softmax"));
    T* logit_grad_data = logit_grad->data<T>();

    const int batch_size = logit_grad->dims()[0];
    const int class_num = logit_grad->dims()[1];
    int block = 512;
    int grid = (batch_size * class_num + block - 1) / block;

    if (context.Attr<bool>("softLabel")) {
      const T* label_data = labels->data<T>();
      SoftCrossEntropyGradientKernel<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              context.device_context())
                              .stream()>>>(logit_grad_data, loss_grad_data,
                                           label_data, batch_size, class_num);
    } else {
      const int* label_data = labels->data<int>();
      CrossEntropyGrad<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              context.device_context())
                              .stream()>>>(logit_grad_data, loss_grad_data,
                                           label_data, batch_size, class_num);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(softmax_with_cross_entropy,
                       ops::SoftmaxWithCrossEntropyCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(softmax_with_cross_entropy_grad,
                       ops::SoftmaxWithCrossEntropyGradCUDAKernel<float>);
