#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include "paddle/framework/op_registry.h"
#include "paddle/platform/cuda_helper.h"
#include "paddle/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

template <typename T>
__global__ void FillFirstRow(T* dist, const int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N + 1) {
    dist[idx] = idx;
  }
}

template <typename T>
__global__ void FillFirstColumn(T* dist, const int M, const int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < M + 1) {
    dist[idx * (N + 1)] = idx;
  }
}

template <typename T>
__global__ void Levenshtein(T* dist, const int* x1, const int* x2, const int M,
                            const int N, const int start) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = N;
  int index = start + idx * offset;
  int row = index / (N + 1);
  int col = index % (N + 1);
  if (row > 0 && col > 0 && row < M + 1 && col < N + 1) {
    int cost = x1[row - 1] == x2[col - 1] ? 0 : 1;
    int dels = dist[(row - 1) * (N + 1) + col] + 1;
    int ins = dist[row * (N + 1) + col - 1] + 1;
    int subs = dist[(row - 1) * (N + 1) + (col - 1)] + cost;
    dist[index] = min(dels, min(ins, subs));
  }
}

template <typename T>
__global__ void SetOutput(T* out, const T* dist, const int M, const int N,
                          bool normalized) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx == 0) {
    out[0] = normalized ? dist[M * (N + 1) + N] / N : dist[M * (N + 1) + N];
  }
}

template <typename Place, typename T>
class EditDistanceGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto* out_t = ctx.Output<framework::Tensor>("Out");

    auto* x1_t = ctx.Input<framework::Tensor>("Hyp");
    auto* x2_t = ctx.Input<framework::Tensor>("Ref");

    out_t->mutable_data<T>(ctx.GetPlace());
    auto out = out_t->data<T>();

    auto normalized = ctx.Attr<bool>("normalized");
    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();

    auto m = x1_t->numel();
    auto n = x2_t->numel();
    T distance = 0.0;
    if (m == 0 || n == 0) {
      distance = std::max(m, n);
      if (normalized) {
        distance = distance / n;
      }
      memory::Copy(boost::get<Place>(ctx.GetPlace()), out, platform::CPUPlace(),
                   &distance, sizeof(T), stream);
    } else {
      framework::Tensor dist_t;
      dist_t.Resize({m + 1, n + 1});
      dist_t.mutable_data<T>(ctx.GetPlace());
      auto dist = dist_t.data<T>();
      auto x1 = x1_t->data<int>();
      auto x2 = x2_t->data<int>();

      FillFirstColumn<T><<<1 + m / PADDLE_CUDA_NUM_THREADS,
                           PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, m, n);

      FillFirstRow<T><<<1 + n / PADDLE_CUDA_NUM_THREADS,
                        PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, n);
      // Compute the elements of distance matrix in the anti-diagonal diretion
      for (int64_t slice = 2; slice < m + n + 1; ++slice) {
        int z_m = slice < m + 1 ? 0 : slice - m;
        int z_n = slice < n + 1 ? 0 : slice - n;
        int size = slice - (z_m + z_n) + 1;  // number of elments in the same
                                             // anti-diagonal line to update
        // the start index at which computes from
        int start = slice < n + 1 ? slice : (z_n + 1) * (n + 1) - 1;
        Levenshtein<T><<<1 + (size - 1) / PADDLE_CUDA_NUM_THREADS,
                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, x1, x2, m,
                                                               n, start);
      }
      SetOutput<T><<<1, 1, 0, stream>>>(out, dist, m, n, normalized);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_CUDA_KERNEL(
    edit_distance,
    ops::EditDistanceGPUKernel<paddle::platform::CUDAPlace, float>);
