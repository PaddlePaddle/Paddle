#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
   http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include "paddle/framework/op_registry.h"
#include "paddle/framework/operator.h"

namespace paddle {
namespace operators {

template <typename T>
struct UniformGenerator {
  T min_, max_;
  unsigned int seed_;

  __host__ __device__ UniformGenerator(T min, T max, int seed)
      : min_(min), max_(max), seed_(seed) {}

  __host__ __device__ T operator()(const unsigned int n) const {
    thrust::minstd_rand rng;
    rng.seed(seed_);
    thrust::uniform_real_distribution<T> dist(min_, max_);
    rng.discard(n);
    return dist(rng);
  }
};

// It seems that Eigen::Tensor::random in GPU will SEGFAULT.
// Use std::random and thrust::random(thrust is a std library in CUDA) to
// implement uniform random.
template <typename T>
class GPUUniformRandomKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* tensor = context.Output<framework::Tensor>("Out");
    T* data = tensor->mutable_data<T>(context.GetPlace());
    unsigned int seed = static_cast<unsigned int>(context.Attr<int>("seed"));
    if (seed == 0) {
      std::random_device rd;
      seed = rd();
    }
    T min = static_cast<T>(context.Attr<float>("min"));
    T max = static_cast<T>(context.Attr<float>("max"));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    int64_t size = tensor->numel();
    thrust::transform(index_sequence_begin, index_sequence_begin + size,
                      thrust::device_ptr<T>(data),
                      UniformGenerator<T>(min, max, seed));
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_GPU_KERNEL(uniform_random,
                       paddle::operators::GPUUniformRandomKernel<float>);
