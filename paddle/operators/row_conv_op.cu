#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/math_function.h"
#include "paddle/operators/row_conv_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

using LoDTensor = framework::LoDTensor;
using framework::Tensor;

namespace {

inline int DivUp(int x, int y) { return (x + y - 1) / y; }

// Forward prop (shared memory version, for small future_context)
template <typename T>
__global__ void RowConvForwardSharedMemory(const T *in, const T *wt,
                                           int num_sequence, int input_dim,
                                           int future_context,
                                           const size_t *batch_indices,
                                           T *out) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int d = blockIdx.x * blx + thx;  // index along input dim

  extern __shared__ T mem[];
  T *sw = mem;

  if (thy < future_context) {
    sw[thy * blx + thx] =
        (d < input_dim) ? wt[thy * input_dim + d] : static_cast<T>(0);
  }
  __syncthreads();

  for (size_t i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < future_context) && ((k + w) < current_timesteps);
           w++) {
        sum += (d < input_dim)
                   ? sw[w * blx + thx] * in[(start + k + w) * input_dim + d]
                   : static_cast<T>(0);
      }
      if (d < input_dim) {
        out[(start + k) * input_dim + d] = sum;
      }
    }
  }
}

// Forward prop (naive version)
template <typename T>
__global__ void RowConvForward(const T *in, const T *wt, int num_sequence,
                               int input_dim, int future_context,
                               const size_t *batch_indices, T *out) {
  int d = blockIdx.x * blockDim.x + threadIdx.x;  // index along input_dim
  int bly = blockDim.y;
  int thy = threadIdx.y;

  if (d >= input_dim) return;

  for (size_t i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < future_context) && ((k + w) < current_timesteps);
           w++) {
        sum += (wt[w * input_dim + d] * in[(start + k + w) * input_dim + d]);
      }
      out[(start + k) * input_dim + d] = sum;
    }
  }
}

// Compute input gradient (shared memory version, for small future_context)
template <typename T>
__global__ void RowConvGradInputSharedMemory(const T *dout, const T *wt,
                                             int num_sequence, int input_dim,
                                             int future_context,
                                             const size_t *batch_indices,
                                             T *din) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int d = blockIdx.x * blx + thx;  // index along input dim

  extern __shared__ T mem[];
  T *sw = mem;
  if (thy < future_context) {
    sw[thy * blx + thx] =
        (d < input_dim) ? wt[thy * input_dim + d] : static_cast<T>(0);
  }
  __syncthreads();

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < future_context) && ((k - w) >= 0); w++) {
        sum += (d < input_dim)
                   ? (sw[w * blx + thx] * dout[(k + start - w) * input_dim + d])
                   : static_cast<T>(0);
      }
      if (d < input_dim) {
        din[(k + start) * input_dim + d] = sum;
      }
    }
  }
}

// Compute input gradient (Naive version)
template <typename T>
__global__ void RowConvGradInput(const T *dout, const T *wt, int num_sequence,
                                 int input_dim, int future_context,
                                 const size_t *batch_indices, T *din) {
  int d = blockIdx.x * blockDim.x + threadIdx.x;  // index along input_dim
  int bly = blockDim.y;
  int thy = threadIdx.y;

  if (d >= input_dim) return;
  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < future_context) && ((k - w) >= 0); w++) {
        sum += (wt[w * input_dim + d] * dout[(k + start - w) * input_dim + d]);
      }
      din[(k + start) * input_dim + d] = sum;
    }
  }
}

// Compute W gradient (small future_context version)
template <typename T>
__global__ void RowConvGradFilterImproved(const T *in, const T *dout,
                                          int num_sequence, int input_dim,
                                          int future_context, int block_x,
                                          int block_y,
                                          const size_t *batch_indices,
                                          T *dfilter) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int gx = blockIdx.x * blx;
  int d = gx + thx;  // index along input dim

  extern __shared__ T mem[];

  int xdim_sh_in = block_y;
  int xdim_sh_dout = block_y;
  // int xdim_sh_dfilter = future_context;
  int ydim_sh_in = block_x;
  int ydim_sh_dout = block_x + future_context - 1;
  int ydim_sh_dfilter = block_y;

  T *sh_in = mem;
  T *sh_dout = &mem[xdim_sh_in * ydim_sh_in];
  T *sh_dfilter = &mem[xdim_sh_in * ydim_sh_in + xdim_sh_dout * ydim_sh_dout];

  if (thy < future_context) {
    sh_dfilter[thy * ydim_sh_dfilter + thx] = static_cast<T>(0);
  }
  __syncthreads();

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    int scaled_cur_steps =
        ((current_timesteps + block_x - 1) / block_x) * block_x;

    for (int k = thy; k < scaled_cur_steps; k += block_x) {
      int pos = start + k;
      sh_in[thx * ydim_sh_in + thy] =
          (d < input_dim && pos < end) ? in[pos * input_dim + d] : T(0);
      sh_dout[thx * ydim_sh_dout + thy + future_context - 1] =
          (d < input_dim && pos < end) ? dout[pos * input_dim + d] : T(0);
      __syncthreads();

      if (thy < future_context - 1) {
        int pos_offset = pos - future_context + 1;
        sh_dout[thx * ydim_sh_dout + thy] =
            (d < input_dim && pos_offset >= start)
                ? dout[pos_offset * input_dim + d]
                : T(0);
      }
      __syncthreads();

      for (int w = 0; w < future_context; w++) {
        T val = sh_in[thy * ydim_sh_in + thx] *
                sh_dout[thy * ydim_sh_dout + thx + future_context - 1 - w];
        __syncthreads();

        for (int offset = 16; offset > 0;
             offset = offset / 2) {  // blockDim.x is 32.
          val += __shfl_down(val, offset);
        }
        __syncthreads();

        if (thx == 0) {
          sh_dfilter[w * ydim_sh_dfilter + thy] += val;
        }
        __syncthreads();
      }
    }
  }
  for (int w = thy; (w < future_context) && (d < input_dim); w += bly) {
    dfilter[w * input_dim + d] += sh_dfilter[w * ydim_sh_dfilter + thx];
  }
}

// Compute weight(filter) gradient
template <typename T>
__global__ void RowConvGradFilter(const T *in, const T *dout, int num_sequence,
                                  int input_dim, int future_context,
                                  int block_x, int block_y,
                                  const size_t *batch_indices, T *dfilter) {
  int blx = blockDim.x;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int gx = blockIdx.x * blx;
  int d = gx + thx;  // index along input dim
  extern __shared__ T mem[];
  T *sh_in = mem;
  T *sh_dout = &mem[block_x * block_y];

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    int scaled_cur_steps =
        ((current_timesteps + block_x - 1) / block_x) * block_x;

    for (int k = thy; k < scaled_cur_steps; k += block_x) {
      int pos = start + k;
      sh_in[thx * block_y + thy] =
          (d < input_dim && pos < end) ? in[pos * input_dim + d] : 0.0;
      __syncthreads();

      for (int w = 0; w < future_context; w++) {
        sh_dout[thx * block_y + thy] =
            (d < input_dim && (k - w) >= 0 && (k - w) < current_timesteps)
                ? dout[(pos - w) * input_dim + d]
                : 0.0;
        __syncthreads();

        T val = sh_in[thy * block_y + thx] * sh_dout[thy * block_y + thx];
        __syncthreads();

        for (int offset = 16; offset > 0;
             offset = offset / 2) {  // blockDim.x is 32.
          val += __shfl_down(val, offset);
        }
        __syncthreads();

        if (thx == 0 && (gx + thy) < input_dim) {
          dfilter[w * input_dim + gx + thy] += val;
        }
      }
    }
  }
}

}  // namespace

template <typename T>
class RowConvKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *X = context.Input<LoDTensor>("X");
    auto *Filter = context.Input<Tensor>("Filter");
    auto *Out = context.Output<LoDTensor>("Out");

    const T *in = X->data<T>();
    const T *weight = Filter->data<T>();
    T *out = Out->mutable_data<T>(context.GetPlace());

    auto batch_indices = X->lod()[0];
    int input_dim = X->dims()[1];
    int num_sequence = batch_indices.size() - 1;
    int future_context = Filter->dims()[0];
    size_t *idx = batch_indices.cuda_data();
    auto stream = context.cuda_device_context().stream();

    if (future_context <= 32) {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      int mem_per_block = (future_context * block_dim.x) * sizeof(T);
      RowConvForwardSharedMemory<
          T><<<grid_dim, block_dim, mem_per_block, stream>>>(
          in, weight, num_sequence, input_dim, future_context, idx, out);
    } else {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      RowConvForward<T><<<grid_dim, block_dim, 0, stream>>>(
          in, weight, num_sequence, input_dim, future_context, idx, out);
    }
  }
};

template <typename T>
class RowConvGradKernel<platform::CUDADeviceContext, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *X = context.Input<LoDTensor>("X");
    auto *Filter = context.Input<Tensor>("Filter");
    auto *dOut = context.Input<LoDTensor>(framework::GradVarName("Out"));
    const T *in = X->data<T>();
    const T *weights = Filter->data<T>();
    const T *dout = dOut->data<T>();

    Tensor *dX = context.Output<LoDTensor>(framework::GradVarName("X"));
    Tensor *dFilter = context.Output<Tensor>(framework::GradVarName("Filter"));

    auto batch_indices = X->lod()[0];
    int input_dim = X->dims()[1];
    int num_sequence = batch_indices.size() - 1;
    int future_context = Filter->dims()[0];
    size_t *idx = batch_indices.cuda_data();

    auto &device_ctx = context.cuda_device_context();
    math::SetConstant<platform::CUDADeviceContext, T> zero;

    if (dFilter) {
      T *dfilter = dFilter->mutable_data<T>(context.GetPlace());
      zero(device_ctx, dFilter, static_cast<T>(0.0));

      if (future_context <= 32) {
        dim3 block_dim = dim3(32, 32);
        dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
        int block_x = block_dim.x;
        int block_y = block_dim.y;
        int mem_per_block =
            (block_y * block_x + block_y * (block_x + future_context - 1) +
             future_context * block_y) *
            sizeof(T);
        RowConvGradFilterImproved<
            T><<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
            in, dout, num_sequence, input_dim, future_context, block_x, block_y,
            idx, dfilter);
      } else {
        dim3 block_dim = dim3(32, 32);
        dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
        int block_x = block_dim.x;
        int block_y = block_dim.y;
        int mem_per_block =
            (block_x * block_y * 2) * sizeof(T);  // For 2 arrays of size 32x32
        RowConvGradFilter<
            T><<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
            in, dout, num_sequence, input_dim, future_context, block_x, block_y,
            idx, dfilter);
      }
    }

    if (dX) {
      T *din = dX->mutable_data<T>(context.GetPlace());
      if (future_context <= 32) {
        dim3 block_dim = dim3(32, 32);
        dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
        int mem_per_block = (future_context * block_dim.x) * sizeof(T);
        RowConvGradInputSharedMemory<
            T><<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
            dout, weights, num_sequence, input_dim, future_context, idx, din);
      } else {
        dim3 block_dim = dim3(32, 32);
        dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
        RowConvGradInput<T><<<grid_dim, block_dim, 0, device_ctx.stream()>>>(
            dout, weights, num_sequence, input_dim, future_context, idx, din);
      }
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    row_conv, ops::RowConvKernel<paddle::platform::CUDADeviceContext, float>);
REGISTER_OP_CUDA_KERNEL(
    row_conv_grad,
    ops::RowConvGradKernel<paddle::platform::CUDADeviceContext, float>);
