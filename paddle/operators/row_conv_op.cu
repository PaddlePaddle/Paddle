#include "hip/hip_runtime.h"
/* Copyright (c) 2017 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/operators/math/math_function.h"
#include "paddle/operators/row_conv_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

using LoDTensor = framework::LoDTensor;
using framework::Tensor;

namespace {

inline int DivUp(int x, int y) { return (x + y - 1) / y; }

// Forward prop (shared memory version, for small context_length)
template <typename T>
__global__ void RowConvForwardSharedMemory(const T *in, const T *wt,
                                           int num_sequence, int input_dim,
                                           int context_length,
                                           const size_t *batch_indices,
                                           T *out) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int d = blockIdx.x * blx + thx;  // index along input dim

  extern __shared__ T mem[];
  T *sw = mem;

  if (thy < context_length) {
    sw[thy * blx + thx] =
        (d < input_dim) ? wt[thy * input_dim + d] : static_cast<T>(0);
  }
  __syncthreads();

  for (size_t i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < context_length) && ((k + w) < current_timesteps);
           w++) {
        sum += (d < input_dim)
                   ? sw[w * blx + thx] * in[(start + k + w) * input_dim + d]
                   : static_cast<T>(0);
      }
      if (d < input_dim) {
        out[(start + k) * input_dim + d] = sum;
      }
    }
  }
}

// Forward prop (naive version)
template <typename T>
__global__ void RowConvForward(const T *in, const T *wt, int num_sequence,
                               int input_dim, int context_length,
                               const size_t *batch_indices, T *out) {
  int d = blockIdx.x * blockDim.x + threadIdx.x;  // index along input_dim
  int bly = blockDim.y;
  int thy = threadIdx.y;

  if (d >= input_dim) return;

  for (size_t i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < context_length) && ((k + w) < current_timesteps);
           w++) {
        sum += (wt[w * input_dim + d] * in[(start + k + w) * input_dim + d]);
      }
      out[(start + k) * input_dim + d] = sum;
    }
  }
}

// Compute input gradient (shared memory version, for small context_length)
template <typename T>
__global__ void RowConvGradInputSharedMemory(const T *dout, const T *wt,
                                             int num_sequence, int input_dim,
                                             int context_length,
                                             const size_t *batch_indices,
                                             T *din) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int d = blockIdx.x * blx + thx;  // index along input dim

  extern __shared__ T mem[];
  T *sw = mem;
  if (thy < context_length) {
    sw[thy * blx + thx] =
        (d < input_dim) ? wt[thy * input_dim + d] : static_cast<T>(0);
  }
  __syncthreads();

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < context_length) && ((k - w) >= 0); w++) {
        sum += (d < input_dim)
                   ? (sw[w * blx + thx] * dout[(k + start - w) * input_dim + d])
                   : static_cast<T>(0);
      }
      if (d < input_dim) {
        din[(k + start) * input_dim + d] = sum;
      }
    }
  }
}

// Compute input gradient (Naive version)
template <typename T>
__global__ void RowConvGradInput(const T *dout, const T *wt, int num_sequence,
                                 int input_dim, int context_length,
                                 const size_t *batch_indices, T *din) {
  int d = blockIdx.x * blockDim.x + threadIdx.x;  // index along input_dim
  int bly = blockDim.y;
  int thy = threadIdx.y;

  if (d >= input_dim) return;
  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    for (int k = thy; k < current_timesteps; k += bly) {
      T sum = 0;
      for (int w = 0; (w < context_length) && ((k - w) >= 0); w++) {
        sum += (wt[w * input_dim + d] * dout[(k + start - w) * input_dim + d]);
      }
      din[(k + start) * input_dim + d] = sum;
    }
  }
}

// Compute weight gradient
template <typename T>
__global__ void RowConvGradFilter(const T *in, const T *dout, int num_sequence,
                                  int input_dim, int context_length,
                                  int block_x, int block_y,
                                  const size_t *batch_indices, T *dfilter) {
  int blx = blockDim.x;
  int bly = blockDim.y;
  int thx = threadIdx.x;
  int thy = threadIdx.y;
  int gx = blockIdx.x * blx;
  int d = gx + thx;  // index along input dim
  extern __shared__ T mem[];
  T *sh_in = mem;
  T *sh_dout = &mem[block_x * block_y];

  //              dweights(w, d) += cur_ip(k + w, d) * cur_dout(k, d);

  for (int i = 0; i < num_sequence; i++) {
    int start = static_cast<int>(batch_indices[i]);
    int end = static_cast<int>(batch_indices[i + 1]);
    int current_timesteps = end - start;
    int scaled_cur_steps =
        ((current_timesteps + block_x - 1) / block_x) * block_x;

    for (int k = thy; k < scaled_cur_steps; k += block_x) {
      int pos = start + k;
      sh_in[thx * block_y + thy] =
          (d < input_dim && pos < end) ? in[pos * input_dim + d] : 0.0;
      __syncthreads();

      for (int w = 0; w < context_length; w++) {
        sh_dout[thx * block_y + thy] =
            (d < input_dim && (k - w) >= 0 && (k - w) < current_timesteps)
                ? dout[(pos - w) * input_dim + d]
                : 0.0;
        __syncthreads();

        T val = sh_in[thy * block_y + thx] * sh_dout[thy * block_y + thx];
        __syncthreads();

        for (int offset = 16; offset > 0;
             offset = offset / 2) {  // blockDim.x is 32.
          val += __shfl_down(val, offset);
        }
        __syncthreads();

        if (thx == 0 && (gx + thy) < input_dim) {
          dfilter[k * input_dim + gx + thy] += val;
        }
      }
    }
  }
}
}  // namespace

template <typename T>
class RowConvKernel<platform::GPUPlace, T> : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *X = context.Input<LoDTensor>("X");
    auto *Filter = context.Input<Tensor>("Filter");
    auto *Out = context.Output<LoDTensor>("Out");

    const T *in = X->data<T>();
    const T *weight = Filter->data<T>();
    T *out = Out->mutable_data<T>(context.GetPlace());

    auto batch_indices = X->lod()[0];
    int input_dim = X->dims()[1];
    int num_sequence = batch_indices.size() - 1;
    int context_length = Filter->dims()[0];
    size_t *idx = batch_indices.data();
    auto stream = context.cuda_device_context().stream();

    if (context_length <= 32) {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      int mem_per_block = (context_length * block_dim.x) * sizeof(T);
      RowConvForwardSharedMemory<
          T><<<grid_dim, block_dim, mem_per_block, stream>>>(
          in, weight, num_sequence, input_dim, context_length, idx, out);
    } else {
      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      RowConvForward<T><<<grid_dim, block_dim, 0, stream>>>(
          in, weight, num_sequence, input_dim, context_length, idx, out);
    }
  }
};

template <typename T>
class RowConvGradKernel<platform::GPUPlace, T> : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    auto *X = context.Input<LoDTensor>("X");
    auto *Filter = context.Input<Tensor>("Filter");
    auto *dOut = context.Input<LoDTensor>(framework::GradVarName("Out"));
    const T *in = X->data<T>();
    const T *weights = Filter->data<T>();
    const T *dout = dOut->data<T>();

    Tensor *dX = context.Output<LoDTensor>(framework::GradVarName("X"));
    Tensor *dFilter = context.Output<Tensor>(framework::GradVarName("Filter"));

    auto batch_indices = X->lod()[0];
    int input_dim = X->dims()[1];
    int num_sequence = batch_indices.size() - 1;
    int context_length = Filter->dims()[0];
    size_t *idx = batch_indices.data();

    auto &device_ctx = context.cuda_device_context();
    math::SetConstant<platform::GPUPlace, T> zero;

    if (dFilter) {
      T *dfilter = dFilter->mutable_data<T>(context.GetPlace());
      zero(device_ctx, dFilter, static_cast<T>(0.0));

      dim3 block_dim = dim3(32, 32);
      dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
      int block_x = block_dim.x;
      int block_y = block_dim.y;
      int mem_per_block =
          (block_x * block_y * 2) * sizeof(T);  // For 2 arrays of size 32x32
      RowConvGradFilter<
          T><<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
          in, dout, num_sequence, input_dim, context_length, block_x, block_y,
          idx, dfilter);
    }

    if (dX) {
      T *din = dX->mutable_data<T>(context.GetPlace());
      if (context_length <= 32) {
        dim3 block_dim = dim3(32, 32);
        dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
        int mem_per_block = (context_length * block_dim.x) * sizeof(T);
        RowConvGradInputSharedMemory<
            T><<<grid_dim, block_dim, mem_per_block, device_ctx.stream()>>>(
            dout, weights, num_sequence, input_dim, context_length, idx, din);
      } else {
        dim3 block_dim = dim3(32, 32);
        dim3 grid_dim = dim3(DivUp(input_dim, block_dim.x), 1);
        RowConvGradInput<T><<<grid_dim, block_dim, 0, device_ctx.stream()>>>(
            dout, weights, num_sequence, input_dim, context_length, idx, din);
      }
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(row_conv,
                       ops::RowConvKernel<paddle::platform::GPUPlace, float>);
REGISTER_OP_GPU_KERNEL(
    row_conv_grad, ops::RowConvGradKernel<paddle::platform::GPUPlace, float>);
