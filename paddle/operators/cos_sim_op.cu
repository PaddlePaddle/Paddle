#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/cos_sim_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void CosSimDyKernel(const T* x_norm, const T* y_norm, const T* x,
                               const T* y, const T* z, const T* dz,
                               const size_t rows, const size_t cols, T* dy) {
  int grid_size = blockDim.x * gridDim.x;
  T y_norm_data = y_norm[0];
  for (int offset = blockIdx.x * blockDim.x + threadIdx.x; offset < rows;
       offset += grid_size) {
    T xy_norm_prod = x_norm[offset] * y_norm_data;
    T dz_data = dz[offset];
    T z_data = z[offset];
    const T* x_data = x + cols * offset;
    T reciprocal_xy_norm_prod = 1 / xy_norm_prod;

    T y_norm_square = y_norm_data * y_norm_data;
    T reciprocal_y_norm_square = 1 / y_norm_square;
    for (size_t i = 0; i < cols; ++i) {
      T dy_data = dz_data * (x_data[i] * reciprocal_xy_norm_prod -
                             z_data * y[i] * reciprocal_y_norm_square);
      platform::CudaAtomicAdd(dy + i, dy_data);
    }
  }
}

template <typename T>
struct CosSimDyFunctor<platform::CUDADeviceContext, T> {
  inline void operator()(const platform::CUDADeviceContext& ctx,
                         const T* x_norm, const T* y_norm, const T* x,
                         const T* y, const T* z, const T* dz, const size_t rows,
                         const size_t cols, T* dy) const {
    const int block_size = 512;
    dim3 threads(block_size, 1);
    dim3 grid(1, (rows + block_size - 1) / block_size);
    CosSimDyKernel<T><<<grid, threads, 0, ctx.stream()>>>(
        x_norm, y_norm, x, y, z, dz, rows, cols, dy);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    cos_sim, ops::CosSimKernel<paddle::platform::CUDADeviceContext, float>);
REGISTER_OP_CUDA_KERNEL(
    cos_sim_grad,
    ops::CosSimGradKernel<paddle::platform::CUDADeviceContext, float>);
