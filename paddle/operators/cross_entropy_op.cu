#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/framework/op_registry.h"
#include "paddle/operators/cross_entropy_op.h"
#include "paddle/platform/assert.h"
#include "paddle/platform/hostdevice.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void CrossEntropyKernel(T* Y, const T* X, const int* label,
                                   const int N, const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < D);
    Y[i] = -tolerable_value(log(X[i * D + label[i]]));
  }
}

template <typename T>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int N, const int D) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    T sum = static_cast<T>(0);
    for (int j = 0; j < D; j++) {
      sum += label[i * D + j] * tolerable_value(log(X[i * D + j]));
    }
    Y[i] = -sum;
  }
}

// TODO(qingqing): make zero setting an common function.
template <typename T>
__global__ void zero(T* X, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    X[i] = 0.0;
  }
}

template <typename T>
__global__ void CrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                           const int* label, const int N,
                                           const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    int idx = i * D + label[i];
    dX[idx] = -dY[i] / X[idx];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                               const T* label, const int N,
                                               const int D) {
  // TOOD(qingqing): optimize for this kernel
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    for (int j = 0; j < D; ++j) {
      int idx = i * D + j;
      dX[idx] = -label[idx] * dY[i] / X[idx];
    }
  }
}

template <typename T>
class CrossEntropyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    auto x = ctx.Input<Tensor>("X");
    auto y = ctx.Output<Tensor>("Y");
    auto label = ctx.Input<Tensor>("Label");

    auto* x_data = x->data<T>();
    y->mutable_data<T>(ctx.GetPlace());
    auto* y_data = y->data<T>();

    int n = x->dims()[0];
    int d = x->dims()[1];
    int block = 512;
    int grid = (n + block - 1) / block;
    // TODO(qingqing) launch kernel on specified stream
    // base on ExecutionContext.
    if (ctx.Attr<int>("soft_label") == 1) {
      auto* label_data = ctx.Input<Tensor>("Label")->data<T>();
      SoftCrossEntropyKernel<T><<<grid, block>>>(y_data, x_data, label_data, n,
                                                 d);
    } else {
      auto* label_data = ctx.Input<Tensor>("Label")->data<int>();
      CrossEntropyKernel<T><<<grid, block>>>(y_data, x_data, label_data, n, d);
    }
  }
};

template <typename T>
class CrossEntropyGradientOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    auto x = ctx.Input<Tensor>("X");
    auto dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto dy = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto label = ctx.Input<Tensor>("Label");

    auto* dx_data = dx->mutable_data<T>(ctx.GetPlace());
    auto* dy_data = dy->data<T>();
    auto* x_data = x->data<T>();

    int n = x->dims()[0];
    int d = x->dims()[1];
    int block = 512;
    int grid = (n * d + block - 1) / block;
    zero<T><<<grid, block>>>(dx_data, n * d);
    grid = (n + block - 1) / block;
    // TODO(qingqing): launch kernel on specified stream
    // base on ExecutionContext.
    if (ctx.Attr<int>("soft_label") == 1) {
      auto* label_data = label->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block>>>(
          dx_data, dy_data, x_data, label_data, n, d);
    } else {
      auto* label_data = label->data<int>();
      CrossEntropyGradientKernel<T><<<grid, block>>>(dx_data, dy_data, x_data,
                                                     label_data, n, d);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(cross_entropy, ops::CrossEntropyOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(cross_entropy_grad,
                       ops::CrossEntropyGradientOpCUDAKernel<float>);
