#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/operators/cross_entropy_op.h"

namespace paddle {
namespace operators {

namespace {

template <typename T>
__global__ void CrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                           const int64_t* label, const int N,
                                           const int D) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    int idx = i * D + label[i];
    dX[idx] = -dY[i] / X[idx];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                               const T* label, const int N,
                                               const int D) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < N * D) {
    int row_ids = ids / D;
    dX[ids] = -label[ids] * dY[row_ids] / X[ids];
  }
}
}  // namespace

template <typename T>
class CrossEntropyOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* label = ctx.Input<Tensor>("Label");
    Tensor* y = ctx.Output<Tensor>("Y");
    y->mutable_data<T>(ctx.GetPlace());

    math::CrossEntropyFunctor<platform::GPUPlace, T>()(
        ctx.device_context(), y, x, label, ctx.Attr<bool>("soft_label"));
  }
};

template <typename T>
class CrossEntropyGradientOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* label = ctx.Input<Tensor>("Label");
    Tensor* dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    dx->mutable_data<T>(ctx.GetPlace());

    const T* dy_data =
        ctx.Input<Tensor>(framework::GradVarName("Y"))->data<T>();
    T* dx_data = dx->mutable_data<T>(ctx.GetPlace());
    const T* x_data = x->data<T>();

    int64_t batch_size = x->dims()[0];
    int64_t class_num = x->dims()[1];

    int block = 512;
    int grid = (batch_size * class_num + block - 1) / block;
    auto stream = ctx.cuda_device_context().stream();

    if (ctx.Attr<bool>("soft_label")) {
      auto* label_data = label->data<T>();
      SoftCrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          dx_data, dy_data, x_data, label_data, batch_size, class_num);
    } else {
      math::SetConstant<platform::GPUPlace, T> functor;
      functor(ctx.device_context(), dx, 0);
      auto* label_data = label->data<int64_t>();
      grid = (batch_size + block - 1) / block;
      CrossEntropyGradientKernel<T><<<grid, block, 0, stream>>>(
          dx_data, dy_data, x_data, label_data, batch_size, class_num);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(cross_entropy, ops::CrossEntropyOpCUDAKernel<float>,
                       ops::CrossEntropyOpCUDAKernel<double>);
REGISTER_OP_GPU_KERNEL(cross_entropy_grad,
                       ops::CrossEntropyGradientOpCUDAKernel<float>,
                       ops::CrossEntropyGradientOpCUDAKernel<double>);
