#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/framework/op_registry.h"
#include "paddle/platform/assert.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__host__ __device__ T clipping_log(const T x) {
  PADDLE_ASSERT(std::is_floating_point<T>::value);
  const T kApproInf = 1e20;
  T v = log(x);
  if (v == INFINITY) {
    return kApproInf;
  }
  if (v == -INFINITY) {
    return -kApproInf;
  }
  return v;
}

template <typename T>
__global__ void CrossEntropyKernel(T* Y, const T* X, const int* label,
                                   const int N, const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < D);
    Y[i] = -clipping_log(X[i * D + label[i]]);
  }
}

// TODO(qingqing): make zero setting an common function.
template <typename T>
__global__ void zero(T* X, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    X[i] = 0.0;
  }
}

template <typename T>
__global__ void CrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                           const int* label, const int N,
                                           const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    int idx = i * D + label[i];
    dX[idx] = -dY[i] / X[idx];
  }
}

template <typename T>
class OnehotCrossEntropyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    auto X = ctx.Input<Tensor>("X");
    const T* Xdata = X->data<T>();
    const int* label_data = ctx.Input<Tensor>("label")->data<int>();
    auto Y = ctx.Output<Tensor>("Y");
    Y->mutable_data<T>(ctx.GetPlace());
    T* Ydata = Y->data<T>();

    int N = X->dims()[0];
    int D = X->dims()[1];
    int block = 512;
    int grid = (N + block - 1) / block;
    // TODO(qingqing) launch kernel on specified stream
    // base on ExecutionContext.
    CrossEntropyKernel<T><<<grid, block>>>(Ydata, Xdata, label_data, N, D);
  }
};

template <typename T>
class OnehotCrossEntropyGradientOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    auto X = ctx.Input<Tensor>("X");
    auto dX = ctx.Output<Tensor>(framework::GradVarName("X"));
    auto dY = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto label = ctx.Input<Tensor>("label");

    auto* dXdata = dX->template mutable_data<T>(ctx.GetPlace());
    auto* dYdata = dY->template data<T>();
    auto* Xdata = X->template data<T>();
    auto* label_data = label->data<int>();

    int N = X->dims()[0];
    int D = X->dims()[1];
    int block = 512;
    int grid = (N * D + block - 1) / block;
    zero<T><<<grid, block>>>(dXdata, N * D);

    grid = (N + block - 1) / block;
    // TODO(qingqing): launch kernel on specified stream
    // base on ExecutionContext.
    CrossEntropyGradientKernel<T><<<grid, block>>>(dXdata, dYdata, Xdata,
                                                   label_data, N, D);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(onehot_cross_entropy,
                       ops::OnehotCrossEntropyOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(onehot_cross_entropy_grad,
                       ops::OnehotCrossEntropyGradientOpCUDAKernel<float>);
