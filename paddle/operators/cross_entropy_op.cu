#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/framework/op_registry.h"
#include "paddle/operators/cross_entropy_op.h"
#include "paddle/platform/assert.h"
#include "paddle/platform/hostdevice.h"

namespace paddle {
namespace operators {

template <typename T>
__global__ void CrossEntropyKernel(T* Y, const T* X, const int* label,
                                   const int N, const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < D);
    Y[i] = -TolerableValue<T>()(log(X[i * D + label[i]]));
  }
}

template <typename T>
__device__ __forceinline__ T sum_single_warp(T val) {
  val += __shfl_down(val, 16);
  val += __shfl_down(val, 8);
  val += __shfl_down(val, 4);
  val += __shfl_down(val, 2);
  val += __shfl_down(val, 1);
  return val;
}

template <typename T>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int class_num) {
  int tid = threadIdx.x;
  extern __shared__ T d_sum[];
  d_sum[tid] = 0;

  int cur_idx = tid;
  int next_idx = blockIdx.x * class_num + tid;
  while (cur_idx < class_num) {
    d_sum[tid] += TolerableValue<T>()(std::log(X[next_idx])) * label[next_idx];
    next_idx += blockDim.x;
    cur_idx += blockDim.x;
  }
  __syncthreads();

  for (unsigned int stride = blockDim.x >> 1; stride >= 32; stride >>= 1) {
    if (tid < stride) d_sum[tid] += d_sum[tid + stride];
    __syncthreads();
  }

  T val = d_sum[tid];
  val = sum_single_warp<T>(val);
  if (tid == 0) Y[blockIdx.x] = -val;
}

// TODO(qingqing): make zero setting a common function.
template <typename T>
__global__ void Zero(T* X, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    X[i] = 0.0;
  }
}

template <typename T>
__global__ void CrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                           const int* label, const int N,
                                           const int D) {
  // TOOD(qingqing) define CUDA_1D_KERNEL_LOOP macro in a common file.
  // CUDA_1D_KERNEL_LOOP(i, N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    int idx = i * D + label[i];
    dX[idx] = -dY[i] / X[idx];
  }
}

template <typename T>
__global__ void SoftCrossEntropyGradientKernel(T* dX, const T* dY, const T* X,
                                               const T* label, const int N,
                                               const int D) {
  int ids = blockIdx.x * blockDim.x + threadIdx.x;
  if (ids < N * D) {
    int row_ids = ids / D;
    dX[ids] = -label[ids] * dY[row_ids] / X[ids];
  }
}

template <typename T>
class CrossEntropyOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* label = ctx.Input<Tensor>("Label");
    Tensor* y = ctx.Output<Tensor>("Y");

    const T* x_data = x->data<T>();
    T* y_data = y->mutable_data<T>(ctx.GetPlace());

    int batch_size = x->dims()[0];
    int class_num = x->dims()[1];

    if (ctx.Attr<bool>("softLabel")) {
      auto* label_data = ctx.Input<Tensor>("Label")->data<T>();
      int block = class_num > 512 ? 512 : pow(2, int(std::log2(class_num)));

      SoftCrossEntropyKernel<
          T><<<batch_size, block, block * sizeof(T),
               reinterpret_cast<const platform::CUDADeviceContext&>(
                   ctx.device_context())
                   .stream()>>>(y_data, x_data, label_data, class_num);
    } else {
      auto* label_data = ctx.Input<Tensor>("Label")->data<int>();
      int block = 512;
      int grid = (batch_size + block - 1) / block;
      CrossEntropyKernel<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              ctx.device_context())
                              .stream()>>>(y_data, x_data, label_data,
                                           batch_size, class_num);
    }
  }
};

template <typename T>
class CrossEntropyGradientOpCUDAKernel : public framework::OpKernel {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");

    const Tensor* x = ctx.Input<Tensor>("X");
    const Tensor* label = ctx.Input<Tensor>("Label");
    Tensor* dx = ctx.Output<Tensor>(framework::GradVarName("X"));

    const T* dy_data =
        ctx.Input<Tensor>(framework::GradVarName("Y"))->data<T>();
    T* dx_data = dx->mutable_data<T>(ctx.GetPlace());
    const T* x_data = x->data<T>();

    int batch_size = x->dims()[0];
    int class_num = x->dims()[1];

    int block = 512;
    int grid = (batch_size * class_num + block - 1) / block;

    if (ctx.Attr<bool>("softLabel")) {
      auto* label_data = label->data<T>();
      SoftCrossEntropyGradientKernel<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              ctx.device_context())
                              .stream()>>>(dx_data, dy_data, x_data, label_data,
                                           batch_size, class_num);
    } else {
      Zero<T><<<grid, block, 0,
                reinterpret_cast<const platform::CUDADeviceContext&>(
                    ctx.device_context())
                    .stream()>>>(dx_data, batch_size * class_num);

      auto* label_data = label->data<int>();
      grid = (batch_size + block - 1) / block;
      CrossEntropyGradientKernel<T><<<
          grid, block, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                              ctx.device_context())
                              .stream()>>>(dx_data, dy_data, x_data, label_data,
                                           batch_size, class_num);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(cross_entropy, ops::CrossEntropyOpCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(cross_entropy_grad,
                       ops::CrossEntropyGradientOpCUDAKernel<float>);
