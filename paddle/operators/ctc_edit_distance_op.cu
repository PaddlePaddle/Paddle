#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include <algorithm>
#include "paddle/framework/op_registry.h"
#include "paddle/platform/cuda_helper.h"
#include "paddle/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

template <typename T>
__global__ void FillFirstRow(T* dist, const int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < N + 1) {
    dist[idx] = idx;
  }
}

template <typename T>
__global__ void FillFirstColumn(T* dist, const int M, const int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < M + 1) {
    dist[idx * (N + 1)] = idx;
  }
}

template <typename T>
__global__ void Levenshtein(T* dist, const T* x1, const T* x2, const int M,
                            const int N, const int start) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = N;
  int index = start + idx * offset;
  int row = index / (N + 1);
  int col = index % (N + 1);
  if (row > 0 && col > 0 && row < M + 1 && col < N + 1) {
    int cost = x1[row - 1] == x2[col - 1] ? 0 : 1;
    int dels = dist[(row - 1) * (N + 1) + col] + 1;
    int ins = dist[row * (N + 1) + col - 1] + 1;
    int subs = dist[(row - 1) * (N + 1) + (col - 1)] + cost;
    dist[index] = min(dels, min(ins, subs));
  }
}

template <typename Place, typename T>
class CTCEditDistanceGPUKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const {
    auto* out_t = ctx.Output<framework::Tensor>("Out");

    auto* x1_t = ctx.Input<framework::Tensor>("X1");
    auto* x2_t = ctx.Input<framework::Tensor>("X2");

    out_t->mutable_data<float>(ctx.GetPlace());

    auto normalized = ctx.Attr<bool>("normalized");
    auto stream = reinterpret_cast<const platform::CUDADeviceContext&>(
                      ctx.device_context())
                      .stream();

    auto m = x1_t->numel();
    auto n = x2_t->numel();
    T distance = 0;
    if (m == 0) {
      distance = n;
    } else if (n == 0) {
      distance = m;
    } else {
      framework::Tensor dist_t;
      dist_t.Resize({m + 1, n + 1});
      dist_t.mutable_data<T>(ctx.GetPlace());
      auto dist = dist_t.data<T>();
      auto x1 = x1_t->data<T>();
      auto x2 = x2_t->data<T>();

      FillFirstColumn<T><<<1 + m / PADDLE_CUDA_NUM_THREADS,
                           PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, m, n);

      FillFirstRow<T><<<1 + n / PADDLE_CUDA_NUM_THREADS,
                        PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, n);
      // compute the elements of distance matrix in the anti-diagonal diretion
      for (size_t slice = 2; slice < m + n + 1; ++slice) {
        int z_m = slice < m + 1 ? 0 : slice - m;
        int z_n = slice < n + 1 ? 0 : slice - n;
        // number of elments in the same anti-diagonal line
        int size = slice - (z_m + z_n) + 1;
        int start = slice < n + 1 ? slice : z_n * (n + 1) - 1;
        Levenshtein<T><<<1 + (size - 1) / PADDLE_CUDA_NUM_THREADS,
                         PADDLE_CUDA_NUM_THREADS, 0, stream>>>(dist, x1, x2, m,
                                                               n, start);
      }

      Place gpu_place = boost::get<Place>(ctx.GetPlace());
      memory::Copy(platform::CPUPlace(), &distance, gpu_place,
                   dist + m * (n + 1) + n, sizeof(T), stream);
    }

    if (normalized) {
      distance = distance / n;
    }
    auto out = out_t->data<float>();
    Place gpu_place = boost::get<Place>(ctx.GetPlace());
    float dist_f = distance;
    memory::Copy(gpu_place, out, platform::CPUPlace(), &dist_f, sizeof(float),
                 stream);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_GPU_KERNEL(
    ctc_edit_distance,
    ops::CTCEditDistanceGPUKernel<paddle::platform::GPUPlace, int>,
    ops::CTCEditDistanceGPUKernel<paddle::platform::GPUPlace, int64_t>);
