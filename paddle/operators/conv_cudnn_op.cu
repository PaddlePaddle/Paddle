#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/framework/eigen.h"
#include "paddle/framework/op_registry.h"
#include "paddle/memory/memory.h"
#include "paddle/operators/conv_op.h"
#include "paddle/platform/assert.h"
#include "paddle/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedFilterDescriptor = platform::ScopedFilterDescriptor;
using ScopedConvolutionDescriptor = platform::ScopedConvolutionDescriptor;
using DataLayout = platform::DataLayout;

static constexpr size_t kCONV_CUDNN_WORKSPACE_LIMIT_BYTES = 1024 * 1024 * 1024;

template <typename T>
class CudnnConvOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");
    auto* input = ctx.Input<Tensor>("Input");
    auto* filter = ctx.Input<Tensor>("Filter");
    auto* output = ctx.Output<Tensor>("Output");

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    int user_workspace_size = ctx.Attr<int>("workspace_size_MB");

    const T* input_data = input->data<T>();
    const T* filter_data = filter->data<T>();
    T* output_data = output->mutable_data<T>(ctx.GetPlace());

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedFilterDescriptor filter_desc;
    ScopedConvolutionDescriptor conv_desc;
    DataLayout layout = DataLayout::kNCHW;

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize2int(input->dims()), groups);
    hipdnnTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
        layout, framework::vectorize2int(output->dims()), groups);
    hipdnnFilterDescriptor_t cudnn_filter_desc = filter_desc.descriptor<T>(
        layout, framework::vectorize2int(filter->dims()), groups);
    hipdnnConvolutionDescriptor_t cudnn_conv_desc =
        conv_desc.descriptor<T>(paddings, strides, dilations);

    int input_channels = input->dims()[1];
    int input_height = input->dims()[2];
    int input_width = input->dims()[3];
    int output_channels = output->dims()[1];
    int output_height = output->dims()[2];
    int output_width = output->dims()[3];

    int group_offset_in = input_channels / groups * input_height * input_width;
    int group_offset_out =
        output_channels / groups * output_height * output_width;
    int group_offset_filter = filter->numel() / groups;
    // ------------------- cudnn conv workspace ---------------------
    void* cudnn_workspace = nullptr;
    size_t workspace_size_in_bytes;  // final workspace to allocate.
    size_t workspace_size_limit = kCONV_CUDNN_WORKSPACE_LIMIT_BYTES;
    if (user_workspace_size > 0) {
      workspace_size_limit = user_workspace_size * 1024 * 1024;
    }
    // ------------------- cudnn conv algorithm ---------------------
    hipdnnConvolutionFwdAlgo_t algo;
    auto handle = ctx.cuda_device_context().cudnn_handle();

    PADDLE_ENFORCE(platform::dynload::hipdnnGetConvolutionForwardAlgorithm(
        handle, cudnn_input_desc, cudnn_filter_desc, cudnn_conv_desc,
        cudnn_output_desc, HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
        workspace_size_limit, &algo));
    // get workspace size able to allocate
    PADDLE_ENFORCE(platform::dynload::hipdnnGetConvolutionForwardWorkspaceSize(
        handle, cudnn_input_desc, cudnn_filter_desc, cudnn_conv_desc,
        cudnn_output_desc, algo, &workspace_size_in_bytes));
    // Allocate on GPU memory
    platform::GPUPlace gpu = boost::get<platform::GPUPlace>(ctx.GetPlace());
    cudnn_workspace = paddle::memory::Alloc(gpu, workspace_size_in_bytes);
    // ------------------- cudnn conv forward ---------------------
    T alpha = 1.0f, beta = 0.0f;
    for (int i = 0; i < groups; i++) {
      PADDLE_ENFORCE(platform::dynload::hipdnnConvolutionForward(
          handle, &alpha, cudnn_input_desc, input_data + i * group_offset_in,
          cudnn_filter_desc, filter_data + i * group_offset_filter,
          cudnn_conv_desc, algo, cudnn_workspace, workspace_size_in_bytes,
          &beta, cudnn_output_desc, output_data + i * group_offset_out));
    }
    // Release the cudnn workspace
    paddle::memory::Free(gpu, cudnn_workspace);
  }
};

template <typename T>
class CudnnConvGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");
    auto input = ctx.Input<Tensor>("Input");
    auto filter = ctx.Input<Tensor>("Filter");
    auto output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto input_grad = ctx.Output<Tensor>(framework::GradVarName("Input"));
    auto filter_grad = ctx.Output<Tensor>(framework::GradVarName("Filter"));

    const T* input_data = input->data<T>();
    const T* output_grad_data = output_grad->data<T>();
    const T* filter_data = filter->data<T>();

    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    std::vector<int> dilations = ctx.Attr<std::vector<int>>("dilations");
    int groups = ctx.Attr<int>("groups");
    int user_workspace_size = ctx.Attr<int>("workspace_size_MB");

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_grad_desc;
    ScopedTensorDescriptor input_grad_desc;

    ScopedFilterDescriptor filter_desc;
    ScopedFilterDescriptor filter_grad_desc;
    ScopedConvolutionDescriptor conv_desc;
    DataLayout layout = DataLayout::kNCHW;

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize2int(input->dims()), groups);
    hipdnnTensorDescriptor_t cudnn_output_grad_desc =
        output_grad_desc.descriptor<T>(
            layout, framework::vectorize2int(output_grad->dims()), groups);
    hipdnnFilterDescriptor_t cudnn_filter_desc = filter_desc.descriptor<T>(
        layout, framework::vectorize2int(filter->dims()), groups);
    hipdnnTensorDescriptor_t cudnn_input_grad_desc = nullptr;
    hipdnnFilterDescriptor_t cudnn_filter_grad_desc = nullptr;

    hipdnnConvolutionDescriptor_t cudnn_conv_desc =
        conv_desc.descriptor<T>(paddings, strides, dilations);

    int input_channels = input->dims()[1];
    int input_height = input->dims()[2];
    int input_width = input->dims()[3];
    int output_grad_channels = filter->dims()[0];
    int output_grad_height = output_grad->dims()[2];
    int output_grad_width = output_grad->dims()[3];

    int group_offset_in = input_channels / groups * input_height * input_width;
    int group_offset_out =
        output_grad_channels / groups * output_grad_height * output_grad_width;
    int group_offset_filter = filter->numel() / groups;
    // ------------------- cudnn backward algorithm ---------------------
    hipdnnConvolutionBwdDataAlgo_t data_algo;
    hipdnnConvolutionBwdFilterAlgo_t filter_algo;
    size_t workspace_size_in_bytes = 0, tmp_size = 0;
    size_t workspace_size_limit = kCONV_CUDNN_WORKSPACE_LIMIT_BYTES;
    if (user_workspace_size > 0) {
      workspace_size_limit = user_workspace_size * 1024 * 1024;
    }

    auto handle = ctx.cuda_device_context().cudnn_handle();
    if (input_grad) {
      cudnn_input_grad_desc = input_grad_desc.descriptor<T>(
          layout, framework::vectorize2int(input_grad->dims()), groups);
      PADDLE_ENFORCE(
          platform::dynload::hipdnnGetConvolutionBackwardDataAlgorithm(
              handle, cudnn_filter_desc,
              // dyDesc: Handle to the previously initialized input differential
              // tensor descriptor.
              cudnn_output_grad_desc, cudnn_conv_desc,
              // dxDesc: Handle to the previously initialized output tensor
              // descriptor.
              cudnn_input_grad_desc,
              HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT,
              workspace_size_limit, &data_algo));
      PADDLE_ENFORCE(
          platform::dynload::hipdnnGetConvolutionBackwardDataWorkspaceSize(
              handle, cudnn_filter_desc, cudnn_output_grad_desc,
              cudnn_conv_desc, cudnn_input_grad_desc, data_algo, &tmp_size));
      workspace_size_in_bytes = std::max(workspace_size_in_bytes, tmp_size);
    }

    if (filter_grad) {
      cudnn_filter_grad_desc = filter_grad_desc.descriptor<T>(
          layout, framework::vectorize2int(filter_grad->dims()), groups);
      PADDLE_ENFORCE(
          platform::dynload::hipdnnGetConvolutionBackwardFilterAlgorithm(
              handle, cudnn_input_desc, cudnn_output_grad_desc, cudnn_conv_desc,
              cudnn_filter_desc,
              HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT,
              workspace_size_limit, &filter_algo));

      PADDLE_ENFORCE(
          platform::dynload::hipdnnGetConvolutionBackwardFilterWorkspaceSize(
              handle, cudnn_input_desc, cudnn_output_grad_desc, cudnn_conv_desc,
              cudnn_filter_desc, filter_algo, &tmp_size));
      workspace_size_in_bytes = std::max(workspace_size_in_bytes, tmp_size);
    }
    // ------------------- cudnn conv workspace ---------------------
    // Already on GPU
    void* cudnn_workspace = nullptr;
    platform::GPUPlace gpu = boost::get<platform::GPUPlace>(ctx.GetPlace());
    cudnn_workspace = paddle::memory::Alloc(gpu, workspace_size_in_bytes);
    // ------------------- cudnn conv backward data ---------------------
    // FIXME(typhoonzero): template type T may not be the same as cudnn call.
    T alpha = 1.0f, beta = 0.0f;
    if (input_grad) {
      T* input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
      auto t = framework::EigenVector<T>::Flatten(*input_grad);
      t.device(ctx.GetEigenDevice<platform::GPUPlace>()) =
          t.constant(static_cast<T>(0));
      for (int i = 0; i < groups; i++) {
        PADDLE_ENFORCE(platform::dynload::hipdnnConvolutionBackwardData(
            handle, &alpha, cudnn_filter_desc,
            filter_data + i * group_offset_filter, cudnn_output_grad_desc,
            output_grad_data + i * group_offset_out, cudnn_conv_desc, data_algo,
            cudnn_workspace, workspace_size_in_bytes, &beta,
            cudnn_input_grad_desc, input_grad_data + i * group_offset_in));
      }
    }
    // ------------------- cudnn conv backward filter ---------------------
    if (filter_grad) {
      T* filter_grad_data = filter_grad->mutable_data<T>(ctx.GetPlace());
      auto t = framework::EigenVector<T>::Flatten(*filter_grad);
      t.device(ctx.GetEigenDevice<platform::GPUPlace>()) =
          t.constant(static_cast<T>(0));
      for (int i = 0; i < groups; i++) {
        PADDLE_ENFORCE(platform::dynload::hipdnnConvolutionBackwardFilter(
            handle, &alpha, cudnn_input_desc, input_data + i * group_offset_in,
            cudnn_output_grad_desc, output_grad_data + i * group_offset_out,
            cudnn_conv_desc, filter_algo, cudnn_workspace,
            workspace_size_in_bytes, &beta, cudnn_filter_grad_desc,
            filter_grad_data + i * group_offset_filter));
      }
    }
    // Release the cudnn workspace
    paddle::memory::Free(gpu, cudnn_workspace);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_GPU_KERNEL(conv_cudnn, paddle::operators::CudnnConvOpKernel<float>);
REGISTER_OP_GPU_KERNEL(conv_cudnn_grad,
                       paddle::operators::CudnnConvGradOpKernel<float>);
