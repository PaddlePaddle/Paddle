#include "hip/hip_runtime.h"
/* Copyright (c) 2017 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/operators/conv_shift_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

using framework::Tensor;

namespace {

inline int div_up(int x, int y) { return (x + y - 1) / y; }

// Some notes on the design:
//
// Each thread is responsible for computing a single output out[k, i].
// Thread blocks are based on tiles of x with height 1 in the batch dimension.
//
// This design is based on the typical use case where the filter
// y is fairly small. For large y, it would probably be more efficient
// to also tile across y.
template <typename T>
__global__ void conv_shift_forward(const T *x, const T *y, T *out, int x_width,
                                   int y_width, int y_half_width,
                                   int batch_size) {
  extern __shared__ T mem[];

  int tx = threadIdx.x;
  int i = blockIdx.x * blockDim.x + tx;  // global x index
  int k = blockIdx.y;                    // batch index

  // Check if we are in a boundary block with fewer x's to process than
  // blockDim.x.
  int num_x =
      (blockIdx.x == gridDim.x - 1) ? (x_width % blockDim.x) : blockDim.x;

  T *sx = mem;
  T *sx_pad = &mem[num_x];
  T *sy = &mem[blockDim.x + y_width];

  // Collaboratively load y[k, :] and length-y padding of x into shared memory.
  int pad_start = blockIdx.x * blockDim.x + num_x + x_width - y_half_width;
  for (int j = tx; j < y_width; j += blockDim.x) {
    sy[j] = y[k * y_width + j];
    sx_pad[j] = x[k * x_width + (pad_start + j) % x_width];
  }

  // Load a cyclically shifted slice of x into shared memory.
  if (tx < num_x) {
    int load_i = (i - y_half_width + x_width) % x_width;
    sx[tx] = x[k * x_width + load_i];
  } else {
    return;
  }
  __syncthreads();

  // Compute dot product of sx[tx:tx + y_width] and sy.
  T sum = 0;
  for (int j = 0; j < y_width; ++j) {
    sum += sx[tx + j] * sy[j];
  }

  // Save to out[k, i].
  out[k * x_width + i] = sum;
}

// Compute x gradient - initial naive implementation with atomic add.
template <typename T>
__global__ void conv_shift_dx(const T *dout, const T *y, T *dx, int x_width,
                              int y_width, int y_half_width, int batch_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // x index
  int j = blockIdx.y;                             // y index
  int k = blockIdx.z;                             // batch index

  if (i < x_width) {
    int index = (i + j - y_half_width + x_width) % x_width;
    atomicAdd(&dx[k * x_width + index],
              dout[k * x_width + i] * y[k * y_width + j]);
  }
}

// Compute y gradient - initial naive implementation with atomic add.
template <typename T>
__global__ void conv_shift_dy(const T *x, const T *dout, T *dy, int x_width,
                              int y_width, int y_half_width, int batch_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // x index
  int j = blockIdx.y;                             // y index
  int k = blockIdx.z;                             // batch index

  if (i < x_width) {
    int index = (i + j - y_half_width + x_width) % x_width;
    atomicAdd(&dy[k * y_width + j],
              x[k * x_width + index] * dout[k * x_width + i]);
  }
}
}  // namespace

template <typename T>
class ConvShiftKernel<platform::GPUPlace, T> : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    const Tensor *X = context.Input<Tensor>("X");
    const Tensor *Y = context.Input<Tensor>("Y");
    Tensor *Out = context.Output<Tensor>("Out");
    const T *x_data = X->data<T>();
    const T *y_data = Y->data<T>();
    T *out_data = Out->mutable_data<T>(context.GetPlace());

    int batch_size = X->dims()[0];
    int x_width = X->dims()[1];
    int y_width = Y->dims()[1];
    int y_half_width = (y_width - 1) / 2;

    const int x_per_block = 256;
    int num_x_blocks = div_up(x_width, x_per_block);
    int mem_per_block = (x_per_block + 2 * y_width) * sizeof(T);

    dim3 grid_dim(num_x_blocks, batch_size);

    auto stream = context.cuda_device_context().stream();

    conv_shift_forward<T><<<grid_dim, x_per_block, mem_per_block, stream>>>(
        x_data, y_data, out_data, x_width, y_width, y_half_width, batch_size);
  }
};

template <typename T>
class ConvShiftGradKernel<platform::GPUPlace, T>
    : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &context) const override {
    const Tensor *X = context.Input<Tensor>("X");
    const Tensor *Y = context.Input<Tensor>("Y");
    const Tensor *dOut = context.Input<Tensor>(framework::GradVarName("Out"));
    const T *x_data = X->data<T>();
    const T *y_data = Y->data<T>();
    const T *dout_data = dOut->data<T>();

    Tensor *dX = context.Output<Tensor>(framework::GradVarName("X"));
    Tensor *dY = context.Output<Tensor>(framework::GradVarName("Y"));

    int batch_size = X->dims()[0];
    int x_width = X->dims()[1];
    int y_width = Y->dims()[1];
    int y_half_width = (y_width - 1) / 2;

    auto stream = context.cuda_device_context().stream();

    const int x_per_block = 256;
    int num_x_blocks = div_up(x_width, x_per_block);
    dim3 grid_dim(num_x_blocks, y_width, batch_size);

    if (dX) {
      T *dx_data = dX->mutable_data<T>(context.GetPlace());
      hipMemsetAsync(dx_data, 0, dX->numel() * sizeof(T), stream);
      conv_shift_dx<T><<<grid_dim, x_per_block, 0, stream>>>(
          dout_data, y_data, dx_data, x_width, y_width, y_half_width,
          batch_size);
    }
    if (dY) {
      T *dy_data = dY->mutable_data<T>(context.GetPlace());
      hipMemsetAsync(dy_data, 0, dY->numel() * sizeof(T), stream);
      conv_shift_dy<T><<<grid_dim, x_per_block, 0, stream>>>(
          x_data, dout_data, dy_data, x_width, y_width, y_half_width,
          batch_size);
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(conv_shift,
                       ops::ConvShiftKernel<paddle::platform::GPUPlace, float>);
REGISTER_OP_GPU_KERNEL(
    conv_shift_grad,
    ops::ConvShiftGradKernel<paddle::platform::GPUPlace, float>);
