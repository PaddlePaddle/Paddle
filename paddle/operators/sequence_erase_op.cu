#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "paddle/operators/sequence_erase_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;
using LoDTensor = framework::LoDTensor;

template <typename T>
__global__ void LabelErasedIdx(const T* in_dat, const int in_len,
                               const T* tokens, const int tokens_len,
                               int* num_erased) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_len) {
    int erased = 0;
    for (int i = 0; i < tokens_len; ++i) {
      if (in_dat[index] == tokens[i]) {
        erased = 1;
      }
    }
    num_erased[index + 1] = erased;
    if (index == 0) {
      num_erased[0] = 0;
    }
  }
}

template <typename T>
__global__ void GetOutLod(const T* num_erased, const int* in_lod,
                          const int lod_len, int* out_lod0) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < lod_len) {
    out_lod0[index] = in_lod[index] - num_erased[in_lod[index]];
  }
}

template <typename T>
__global__ void SetOutput(const T* in_dat, const int in_len,
                          const int* num_erased, T* out_dat) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_len) {
    if (in_dat[index] != in_dat[index + 1]) {
      out_dat[index - num_erased[index]] = in_dat[index];
    }
  }
}

template <typename T>
class SequenceEraseOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<LoDTensor>("X");
    auto* out = ctx.Output<LoDTensor>("Out");

    auto lod = in->lod();
    PADDLE_ENFORCE_EQ(lod.size(), 1UL, "Only support one level sequence now.");
    auto tokens = ctx.Attr<std::vector<T>>("tokens");
    auto tokens_len = tokens.size();
    auto in_len = in->numel();
    auto in_dat = in->data<T>();
    auto lod0 = lod[0];

    thrust::host_vector<T> host_tokens(tokens_len);
    for (size_t i = 0; i < tokens.size(); ++i) {
      host_tokens[i] = tokens[i];
    }
    thrust::device_vector<T> dev_tokens = host_tokens;
    thrust::device_vector<int> num_erased(in_len + 1);

    T* dev_tokens_ptr = thrust::raw_pointer_cast(dev_tokens.data());
    int* num_erased_ptr = thrust::raw_pointer_cast(num_erased.data());

    auto stream = ctx.cuda_device_context().stream();
    LabelErasedIdx<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                     PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_dat, in_len, dev_tokens_ptr, tokens_len, num_erased_ptr);
    thrust::inclusive_scan(num_erased.begin() + 1, num_erased.end(),
                           num_erased.begin() + 1);

    // Calc LoD
    auto lod_len = lod0.size();
    thrust::host_vector<int> host_lod(lod_len);
    for (size_t i = 0; i < lod_len; ++i) {
      host_lod[i] = lod0[i];
    }
    thrust::device_vector<int> dev_in_lod = host_lod;
    thrust::device_vector<int> dev_out_lod(lod_len);
    int* dev_in_lod_ptr = thrust::raw_pointer_cast(dev_in_lod.data());
    int* dev_out_lod_ptr = thrust::raw_pointer_cast(dev_out_lod.data());
    GetOutLod<<<(lod_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        num_erased_ptr, dev_in_lod_ptr, lod_len, dev_out_lod_ptr);
    thrust::host_vector<int> host_out_lod = dev_out_lod;
    std::vector<int> out_lod0(lod_len, 0);
    for (size_t i = 0; i < lod_len; i++) {
      out_lod0[i] = host_out_lod[i];
    }
    framework::LoD out_lod;
    out_lod.push_back(out_lod0);
    out->set_lod(out_lod);

    // Set output
    out->Resize({out_lod0.back(), 1});
    auto out_dat = out->mutable_data<T>(ctx.GetPlace());
    SetOutput<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_dat, in_len,
                                                      num_erased_ptr, out_dat);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(sequence_erase,
                        paddle::operators::SequenceEraseOpCUDAKernel<int32_t>);
