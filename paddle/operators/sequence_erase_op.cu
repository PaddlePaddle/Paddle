#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "paddle/operators/sequence_erase_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;
using LoDTensor = framework::LoDTensor;

template <typename T>
__global__ void LabelErasedIdx(const T* in_dat, const int64_t in_len,
                               const int* tokens, const size_t tokens_len,
                               size_t* num_erased) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_len) {
    for (size_t i = 0; i < tokens_len; ++i) {
      if (in_dat[index] == tokens[i]) {
        num_erased[index + 1] = 1;
        break;
      }
    }
  }
}

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod,
                          const size_t lod_len, size_t* out_lod0) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < lod_len) {
    out_lod0[index] = in_lod[index] - num_erased[in_lod[index]];
  }
}

template <typename T>
__global__ void SetOutput(const T* in_dat, const int64_t in_len,
                          const size_t* num_erased, T* out_dat) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < in_len) {
    if (num_erased[index] == num_erased[index + 1]) {
      out_dat[index - num_erased[index]] = in_dat[index];
    }
  }
}

template <typename T>
class SequenceEraseOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in = ctx.Input<LoDTensor>("X");
    auto* out = ctx.Output<LoDTensor>("Out");

    auto lod = in->lod();
    PADDLE_ENFORCE_EQ(lod.size(), 1UL, "Only support one level sequence now.");
    PADDLE_ENFORCE_EQ(lod[0].back(), (size_t)in->numel(),
                      "The actual size mismatches with the LoD information.");
    auto tokens = ctx.Attr<std::vector<int>>("tokens");
    auto in_len = in->numel();
    auto in_dat = in->data<T>();
    // Copy tokens to GPU
    thrust::device_vector<int> dev_tokens(tokens.begin(), tokens.end());
    int* dev_tokens_ptr = thrust::raw_pointer_cast(dev_tokens.data());

    // Count number of elements to be erased
    thrust::device_vector<size_t> num_erased(in_len + 1, 0);
    size_t* num_erased_ptr = thrust::raw_pointer_cast(num_erased.data());
    auto stream = ctx.cuda_device_context().stream();
    LabelErasedIdx<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                     PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        in_dat, in_len, dev_tokens_ptr, tokens.size(), num_erased_ptr);
    thrust::inclusive_scan(num_erased.begin() + 1, num_erased.end(),
                           num_erased.begin() + 1);

    // Copy LoD to GPU
    auto lod0 = lod[0];
    auto lod_len = lod0.size();
    thrust::device_vector<size_t> dev_in_lod = lod0;
    size_t* dev_in_lod_ptr = thrust::raw_pointer_cast(dev_in_lod.data());

    // Calc output LoD
    thrust::device_vector<size_t> dev_out_lod(lod_len);
    size_t* dev_out_lod_ptr = thrust::raw_pointer_cast(dev_out_lod.data());
    GetOutLod<<<(lod_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
        num_erased_ptr, dev_in_lod_ptr, lod_len, dev_out_lod_ptr);
    // Set LoD for output
    std::vector<size_t> out_lod0(dev_out_lod.begin(), dev_out_lod.end());
    framework::LoD out_lod;
    out_lod.push_back(out_lod0);
    out->set_lod(out_lod);

    // Set output
    out->Resize({static_cast<int64_t>(out_lod0.back()), 1});
    auto out_dat = out->mutable_data<T>(ctx.GetPlace());
    SetOutput<<<(in_len - 1) / PADDLE_CUDA_NUM_THREADS + 1,
                PADDLE_CUDA_NUM_THREADS, 0, stream>>>(in_dat, in_len,
                                                      num_erased_ptr, out_dat);
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(sequence_erase,
                        paddle::operators::SequenceEraseOpCUDAKernel<int32_t>,
                        paddle::operators::SequenceEraseOpCUDAKernel<int64_t>);
