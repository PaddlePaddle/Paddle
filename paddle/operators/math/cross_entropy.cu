#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/operators/math/cross_entropy.h"

namespace paddle {
namespace operators {
namespace math {

namespace {
template <typename T>
__global__ void CrossEntropyKernel(T* Y, const T* X, const int64_t* label,
                                   const int N, const int D) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    PADDLE_ASSERT(label[i] >= 0 && label[i] < D);
    Y[i] = -math::TolerableValue<T>()(log(X[i * D + label[i]]));
  }
}

template <typename T>
__device__ __forceinline__ T sum_single_warp(T val) {
  val += __shfl_down(val, 16);
  val += __shfl_down(val, 8);
  val += __shfl_down(val, 4);
  val += __shfl_down(val, 2);
  val += __shfl_down(val, 1);
  return val;
}

// CUDA do not support dynamic arrary in template
// https://stackoverflow.com/questions/20497209
template <typename T>
struct SharedMemory {
  // Ensure that we won't compile any un-specialized types
  __device__ T* GetPointer() { return NULL; }
};

template <>
struct SharedMemory<float> {
  __device__ float* GetPointer() {
    extern __shared__ float s_float[];
    return s_float;
  }
};

template <>
struct SharedMemory<double> {
  __device__ double* GetPointer() {
    extern __shared__ double s_double[];
    return s_double;
  }
};

template <typename T>
__global__ void SoftCrossEntropyKernel(T* Y, const T* X, const T* label,
                                       const int class_num) {
  int tid = threadIdx.x;
  SharedMemory<T> d_sum_shared;
  T* d_sum = d_sum_shared.GetPointer();
  d_sum[tid] = 0;

  int cur_idx = tid;
  int next_idx = blockIdx.x * class_num + tid;
  while (cur_idx < class_num) {
    d_sum[tid] +=
        math::TolerableValue<T>()(std::log(X[next_idx])) * label[next_idx];
    next_idx += blockDim.x;
    cur_idx += blockDim.x;
  }
  __syncthreads();

  for (unsigned int stride = blockDim.x >> 1; stride >= 32; stride >>= 1) {
    if (tid < stride) d_sum[tid] += d_sum[tid + stride];
    __syncthreads();
  }

  T val = d_sum[tid];
  val = sum_single_warp<T>(val);
  if (tid == 0) Y[blockIdx.x] = -val;
}
}  // namespace

using Tensor = framework::Tensor;

template <typename T>
class CrossEntropyFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& ctx, framework::Tensor* out,
                  const framework::Tensor* prob,
                  const framework::Tensor* labels, bool softLabel) {
    const T* prob_data = prob->data<T>();
    T* loss_data = out->mutable_data<T>(ctx.GetPlace());

    int batch_size = prob->dims()[0];
    int class_num = prob->dims()[1];

    if (softLabel) {
      const T* label_data = labels->data<T>();
      int block = class_num > 512 ? 512 : pow(2, int(std::log2(class_num)));

      SoftCrossEntropyKernel<T><<<
          batch_size, block, block * sizeof(T),
          reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
          loss_data, prob_data, label_data, class_num);
    } else {
      const int64_t* label_data = labels->data<int64_t>();
      int block = 512;
      int grid = (batch_size + block - 1) / block;
      CrossEntropyKernel<T><<<
          grid, block, 0,
          reinterpret_cast<const platform::CUDADeviceContext&>(ctx).stream()>>>(
          loss_data, prob_data, label_data, batch_size, class_num);
    }
  }
};

template class CrossEntropyFunctor<platform::GPUPlace, float>;
template class CrossEntropyFunctor<platform::GPUPlace, double>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
