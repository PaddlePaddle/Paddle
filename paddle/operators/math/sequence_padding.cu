#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/sequence_padding.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T, bool NormByTimes, bool Padding>
__global__ void SequencePaddingKernel(T* padding, T* sequence,
                                      const size_t* sequence_start_positions,
                                      const size_t sequence_width,
                                      const size_t max_sequence_length,
                                      const size_t num_sequences) {
  size_t padding_idx = blockIdx.y;
  size_t start_pos = sequence_start_positions[padding_idx];
  size_t sequence_length =
      sequence_start_positions[padding_idx + 1] - start_pos;

  size_t sequence_idx = blockIdx.x * blockDim.y + threadIdx.y;
  size_t padding_base_idx =
      (sequence_idx * num_sequences + padding_idx) * sequence_width;
  size_t sequence_base_idx = (start_pos + sequence_idx) * sequence_width;

  if (sequence_idx < sequence_length) {
    T scale = NormByTimes ? (1.0f / static_cast<T>(sequence_length)) : 1.0f;
    if (Padding) {
      /* sequence -> padding */
      for (size_t i = threadIdx.x; i < sequence_width; i += blockDim.x) {
        padding[padding_base_idx + i] = scale * sequence[sequence_base_idx + i];
      }
    } else {
      /* padding -> sequence */
      for (size_t i = threadIdx.x; i < sequence_width; i += blockDim.x) {
        sequence[sequence_base_idx + i] = scale * padding[padding_base_idx + i];
      }
    }
  } else if (sequence_idx < max_sequence_length) {
    if (Padding) {
      /* sequence -> padding */
      for (size_t i = threadIdx.x; i < sequence_width; i += blockDim.x) {
        padding[padding_base_idx + i] = 0;
      }
    }
  }
}

template <typename T>
class PaddingLoDTensorFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::LoDTensor& seq, framework::Tensor& padding,
                  bool norm_by_times) {
    auto lod = seq.lod();
    PADDLE_ENFORCE_GT(lod.size(), 0UL,
                      "The lod of LoDTensor seq should not be null.");

    const size_t level = 0;
    framework::LoD abs_offset_lod = framework::ToAbsOffset(lod);

    auto seq_dims = seq.dims();
    PADDLE_ENFORCE_EQ(seq_dims[0],
                      static_cast<int64_t>(abs_offset_lod[level].back()),
                      "The first dimension of LoDTensor seq should be "
                      "equal to the sum of all sequences's length.");

    auto padding_dims = padding.dims();
    PADDLE_ENFORCE_EQ(padding_dims.size(), 3UL,
                      "The input padding should be a 3-D Tensor of shape "
                      "[max_sequence_length, num_sequences, sequence_width].");

    int64_t max_sequence_length = MaximumSequenceLength(lod, level);
    PADDLE_ENFORCE_EQ(padding_dims[0], max_sequence_length,
                      "The first dimension of Tensor padding should be the "
                      "maximum length of all sequences in LoDTensor seq.");

    const int64_t num_sequences = abs_offset_lod[level].size() - 1;
    PADDLE_ENFORCE_EQ(padding_dims[1], num_sequences,
                      "The second dimension of Tensor padding should be the "
                      "number of sequences in LoDTensor seq.");

    const int64_t sequence_width = seq.numel() / seq_dims[0];
    PADDLE_ENFORCE_EQ(padding_dims[2], sequence_width,
                      "The third dimension of Tensor padding should be the "
                      "width of sequence in LoDTensor seq.");

    if (!norm_by_times && num_sequences == 1UL) {
      Copy(seq, context.GetPlace(), context, &padding);
      padding.Resize(padding_dims);
      return;
    }

    const int64_t kBlockSize = 512;

    /* At least use 32 threads to copy sequence_width elements,
     * and at least 8 elements for each thread.
     */
    size_t block_dim_x =
        std::min(((((sequence_width + 7) >> 3) + 31) >> 5) << 5, kBlockSize);
    size_t block_dim_y = kBlockSize / block_dim_x;
    dim3 threads(block_dim_x, block_dim_y);

    size_t grid_dim_x = (max_sequence_length + block_dim_y - 1) / block_dim_y;
    size_t grid_dim_y = num_sequences;
    dim3 grid(grid_dim_x, grid_dim_y);

    const T* seq_data = seq.data<T>();
    T* padding_data = padding.data<T>();
    if (norm_by_times) {
      SequencePaddingKernel<T, 1, 1><<<grid, threads, 0, context.stream()>>>(
          padding_data, const_cast<T*>(seq_data),
          abs_offset_lod[level].cuda_data(), sequence_width,
          max_sequence_length, num_sequences);
    } else {
      SequencePaddingKernel<T, 0, 1><<<grid, threads, 0, context.stream()>>>(
          padding_data, const_cast<T*>(seq_data),
          abs_offset_lod[level].cuda_data(), sequence_width,
          max_sequence_length, num_sequences);
    }
  }
};

template <typename T>
class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  framework::LoDTensor& seq, const framework::Tensor& padding,
                  bool norm_by_times) {
    auto lod = seq.lod();
    PADDLE_ENFORCE_GT(lod.size(), 0UL,
                      "The lod of LoDTensor seq should not be null.");

    const size_t level = 0;
    framework::LoD abs_offset_lod = framework::ToAbsOffset(lod);

    auto seq_dims = seq.dims();
    PADDLE_ENFORCE_EQ(seq_dims[0],
                      static_cast<int64_t>(abs_offset_lod[level].back()),
                      "The first dimension of LoDTensor seq should be "
                      "equal to the sum of all sequences's length.");

    auto padding_dims = padding.dims();
    PADDLE_ENFORCE_EQ(padding_dims.size(), 3UL,
                      "The input padding should be a 3-D Tensor of shape "
                      "[max_sequnece_length, num_sequences, sequence_width].");

    int64_t max_sequence_length = MaximumSequenceLength(lod, level);
    PADDLE_ENFORCE_EQ(padding_dims[0], max_sequence_length,
                      "The first dimension of Tensor padding should be "
                      "the maximum length of all sequences in LoDTensor seq.");

    const int64_t num_sequences = abs_offset_lod[level].size() - 1;
    PADDLE_ENFORCE_EQ(padding_dims[1], num_sequences,
                      "The second dimension of Tensor padding should be "
                      "the number of sequences in LoDTensor seq.");

    const int64_t sequence_width = seq.numel() / seq_dims[0];
    PADDLE_ENFORCE_EQ(padding_dims[2], sequence_width,
                      "The third dimension of Tensor padding should be the "
                      "width of sequence in LoDTensor seq.");

    if (!norm_by_times && num_sequences == 1UL) {
      Copy(padding, context.GetPlace(), context, &seq);
      seq.Resize(seq_dims);
      return;
    }

    const int64_t kBlockSize = 512;

    /* At least use 32 threads to copy sequence_width elements,
     * and at least 8 elements for each thread.
     */
    size_t block_dim_x =
        std::min(((((sequence_width + 7) >> 3) + 31) >> 5) << 5, kBlockSize);
    size_t block_dim_y = kBlockSize / block_dim_x;
    dim3 threads(block_dim_x, block_dim_y);

    size_t grid_dim_x = (max_sequence_length + block_dim_y - 1) / block_dim_y;
    size_t grid_dim_y = num_sequences;
    dim3 grid(grid_dim_x, grid_dim_y);

    const T* padding_data = padding.data<T>();
    T* seq_data = seq.data<T>();
    if (norm_by_times) {
      SequencePaddingKernel<T, 1, 0><<<grid, threads, 0, context.stream()>>>(
          const_cast<T*>(padding_data), seq_data,
          abs_offset_lod[level].cuda_data(), sequence_width,
          max_sequence_length, num_sequences);
    } else {
      SequencePaddingKernel<T, 0, 0><<<grid, threads, 0, context.stream()>>>(
          const_cast<T*>(padding_data), seq_data,
          abs_offset_lod[level].cuda_data(), sequence_width,
          max_sequence_length, num_sequences);
    }
  }
};

template class PaddingLoDTensorFunctor<platform::CUDADeviceContext, float>;
template class UnpaddingLoDTensorFunctor<platform::CUDADeviceContext, float>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
