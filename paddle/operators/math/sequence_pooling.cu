#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/math_function.h"
#include "paddle/operators/math/sequence_pooling.h"

namespace paddle {
namespace operators {
namespace math {

#define FLT_MAX __FLT_MAX__

template <typename T>
__global__ void KeMaxSequencePool(const T* input, const size_t* starts,
                                  T* output, int* index, int64_t num_seq,
                                  int64_t dim) {
  int dim_idx = threadIdx.x;
  int seq_id = blockIdx.x;
  if (seq_id >= num_seq) return;
  size_t start = starts[seq_id];
  size_t end = starts[seq_id + 1];

  for (int64_t i = dim_idx; i < dim; i += blockDim.x) {
    T max_val = static_cast<T>(-FLT_MAX);
    int max_id = -1;
    for (size_t step_id = start; step_id < end; step_id++) {
      if (max_val < input[step_id * dim + i]) {
        max_val = input[step_id * dim + i];
        max_id = step_id;
      }
    }
    output[seq_id * dim + i] = max_val;
    index[seq_id * dim + i] = max_id;
  }
}

template <typename T>
class MaxSeqPoolFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::LoDTensor& input, framework::Tensor* output,
                  framework::Tensor* index) {
    auto in_dims = input.dims();
    auto out_dims = output->dims();
    auto idx_dims = index->dims();
    PADDLE_ENFORCE_GT(in_dims.size(), static_cast<int64_t>(1));
    PADDLE_ENFORCE_GT(out_dims.size(), 1);
    for (int64_t i = 1; i < in_dims.size(); ++i) {
      PADDLE_ENFORCE_EQ(in_dims[i], out_dims[i]);
    }
    PADDLE_ENFORCE_EQ(idx_dims, out_dims);

    auto starts = input.lod()[0];
    const T* in_data = input.data<T>();
    T* out_data = output->data<T>();
    int* max_index = index->data<int>();

    int64_t num_seq = out_dims[0];
    int64_t dim = output->numel() / num_seq;

    dim3 threads(256, 1);
    dim3 grid(num_seq, 1);
    auto stream = context.stream();
    KeMaxSequencePool<T><<<grid, threads, 0, stream>>>(
        in_data, starts.cuda_data(), out_data, max_index, num_seq, dim);
  }
};

template <typename T>
__global__ void KeMaxSequencePoolGrad(const T* out_grad, const int* max_index,
                                      T* in_grad, int64_t num_seq,
                                      int64_t dim) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int col_idx = idx % dim;
  if (idx < num_seq * dim) {
    int step_id = max_index[idx];
    in_grad[step_id * dim + col_idx] = out_grad[idx];
  }
}

template <typename T>
class MaxSeqPoolGradFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& out_grad,
                  const framework::Tensor& index,
                  framework::LoDTensor* in_grad) {
    auto og_dims = out_grad.dims();
    auto idx_dims = index.dims();
    auto ig_dims = in_grad->dims();
    PADDLE_ENFORCE_GT(og_dims.size(), static_cast<int64_t>(1));
    PADDLE_ENFORCE_GT(ig_dims.size(), static_cast<int64_t>(1));
    for (int64_t i = 1; i < og_dims.size(); ++i) {
      PADDLE_ENFORCE_EQ(og_dims[i], ig_dims[i]);
    }
    PADDLE_ENFORCE_EQ(idx_dims, og_dims);

    const T* og_data = out_grad.data<T>();
    const int* max_index = index.data<int>();
    T* ig_data = in_grad->data<T>();

    SetConstant<platform::CUDADeviceContext, T> set_zero;
    set_zero(context, in_grad, static_cast<T>(0.0));
    int64_t num_seq = og_dims[0];
    int64_t dim = out_grad.numel() / num_seq;

    unsigned int blocks = (num_seq * dim + 128 - 1) / 128;
    dim3 threads(128, 1);
    dim3 grid(blocks, 1);
    auto stream = context.stream();
    KeMaxSequencePoolGrad<T><<<grid, threads, 0, stream>>>(
        og_data, max_index, ig_data, num_seq, dim);
  }
};

template class MaxSeqPoolFunctor<platform::CUDADeviceContext, float>;
template class MaxSeqPoolFunctor<platform::CUDADeviceContext, double>;
template class MaxSeqPoolGradFunctor<platform::CUDADeviceContext, float>;
template class MaxSeqPoolGradFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
