#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/math/sequence2batch.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void CopyMatrixRowsKernel(const T* src, T* dst, const size_t* index,
                                     int64_t height, int64_t width,
                                     bool is_src_index) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id = blockIdx.x + idy * GridDimX;
  while (id < height) {
    int src_idx = is_src_index ? index[id] : id;
    int dst_idx = is_src_index ? id : index[id];
    const T* src_data = src + src_idx * width;
    T* dst_data = dst + dst_idx * width;
    for (int i = idx; i < width; i += BlockDimX) {
      dst_data[i] = src_data[i];
    }
    id += BlockDimY * GridDimX;
  }
}

template <typename T>
class CopyMatrixRowsFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& src,
                  framework::Vector<size_t> index_lod, framework::Tensor& dst,
                  bool is_src_index) {
    size_t* index = index_lod.cuda_data();
    auto src_dims = src.dims();
    auto dst_dims = dst.dims();
    PADDLE_ENFORCE_EQ(src_dims.size(), 2,
                      "The src must be matrix with rank 2.");
    PADDLE_ENFORCE_EQ(dst_dims.size(), 2,
                      "The dst must be matrix with rank 2.");
    PADDLE_ENFORCE_EQ(src_dims[1], dst_dims[1],
                      "The width of src and dst must be same.");
    auto height = dst_dims[0];
    auto width = dst_dims[1];
    auto* src_data = src.data<T>();
    auto* dst_data = dst.data<T>();

    dim3 threads(128, 8);
    dim3 grid(8, 1);
    auto stream = context.stream();
    CopyMatrixRowsKernel<T, 128, 8, 8><<<grid, threads, 0, stream>>>(
        src_data, dst_data, index, height, width, is_src_index);
  }
};

template class CopyMatrixRowsFunctor<platform::CUDADeviceContext, float>;
template class CopyMatrixRowsFunctor<platform::CUDADeviceContext, double>;

template class LoDTensor2BatchFunctor<platform::CUDADeviceContext, float>;
template class LoDTensor2BatchFunctor<platform::CUDADeviceContext, double>;
template class Batch2LoDTensorFunctor<platform::CUDADeviceContext, float>;
template class Batch2LoDTensorFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
