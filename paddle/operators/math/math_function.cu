/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/math_function.h"

namespace paddle {
namespace operators {
namespace math {

template <>
void gemm<platform::GPUPlace, float>(const CBLAS_TRANSPOSE transA,
                                     const CBLAS_TRANSPOSE transB, const int M,
                                     const int N, const int K,
                                     const float alpha, const float* A,
                                     const float* B, const float beta, float* C,
                                     platform::DeviceContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      reinterpret_cast<platform::CUDADeviceContext*>(context)->cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gemm<platform::GPUPlace, double>(const CBLAS_TRANSPOSE transA,
                                      const CBLAS_TRANSPOSE transB, const int M,
                                      const int N, const int K,
                                      const double alpha, const double* A,
                                      const double* B, const double beta,
                                      double* C,
                                      platform::DeviceContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      reinterpret_cast<platform::CUDADeviceContext*>(context)->cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void matmul<platform::GPUPlace, float>(const framework::LODTensor& matrix_a,
                                       bool trans_a,
                                       const framework::LODTensor& matrix_b,
                                       bool trans_b, float alpha,
                                       framework::LODTensor* matrix_out,
                                       float beta,
                                       platform::DeviceContext* context) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in GPUPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::GPUPlace, float>(
      transA, transB, M, N, K, alpha, matrix_a.data<float>(),
      matrix_b.data<float>(), beta, matrix_out->data<float>(), context);
}

template <>
void matmul<platform::GPUPlace, double>(const framework::LODTensor& matrix_a,
                                        bool trans_a,
                                        const framework::LODTensor& matrix_b,
                                        bool trans_b, double alpha,
                                        framework::LODTensor* matrix_out,
                                        double beta,
                                        platform::DeviceContext* context) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in GPUPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::GPUPlace, double>(
      transA, transB, M, N, K, alpha, matrix_a.data<double>(),
      matrix_b.data<double>(), beta, matrix_out->data<double>(), context);
}

}  // namespace math
}  // namespace operators
}  // namespace paddle
