/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/framework/data_type.h"
#include "paddle/operators/math/math_function.h"
#include "paddle/operators/math/math_function_impl.h"

namespace paddle {
namespace operators {
namespace math {

template <>
void gemm<platform::GPUPlace, float>(const platform::DeviceContext& context,
                                     const CBLAS_TRANSPOSE transA,
                                     const CBLAS_TRANSPOSE transB, const int M,
                                     const int N, const int K,
                                     const float alpha, const float* A,
                                     const float* B, const float beta,
                                     float* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gemm<platform::GPUPlace, double>(const platform::DeviceContext& context,
                                      const CBLAS_TRANSPOSE transA,
                                      const CBLAS_TRANSPOSE transB, const int M,
                                      const int N, const int K,
                                      const double alpha, const double* A,
                                      const double* B, const double beta,
                                      double* C) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gemm<platform::GPUPlace, float>(const platform::DeviceContext& context,
                                     const bool transA, const bool transB,
                                     const int M, const int N, const int K,
                                     const float alpha, const float* A,
                                     const int lda, const float* B,
                                     const int ldb, const float beta, float* C,
                                     const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasSgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));
}

template <>
void gemm<platform::GPUPlace, double>(const platform::DeviceContext& context,
                                      const bool transA, const bool transB,
                                      const int M, const int N, const int K,
                                      const double alpha, const double* A,
                                      const int lda, const double* B,
                                      const int ldb, const double beta,
                                      double* C, const int ldc) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA = transA == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB = transB == false ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemm(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));
}

template <>
void matmul<platform::GPUPlace, float>(
    const platform::DeviceContext& context, const framework::Tensor& matrix_a,
    bool trans_a, const framework::Tensor& matrix_b, bool trans_b, float alpha,
    framework::Tensor* matrix_out, float beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in GPUPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::GPUPlace, float>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<float>(),
      matrix_b.data<float>(), beta, matrix_out->data<float>());
}

template <>
void matmul<platform::GPUPlace, double>(
    const platform::DeviceContext& context, const framework::Tensor& matrix_a,
    bool trans_a, const framework::Tensor& matrix_b, bool trans_b, double alpha,
    framework::Tensor* matrix_out, double beta) {
  auto dim_a = matrix_a.dims();
  auto dim_b = matrix_b.dims();
  auto dim_out = matrix_out->dims();
  PADDLE_ENFORCE(dim_a.size() == 2 && dim_b.size() == 2 && dim_out.size() == 2,
                 "The input and output of matmul be matrix");

  PADDLE_ENFORCE(platform::is_gpu_place(matrix_a.place()) &&
                     platform::is_gpu_place(matrix_b.place()) &&
                     platform::is_gpu_place(matrix_out->place()),
                 "Matrix must all be in GPUPlace");

  int M = dim_out[0];
  int N = dim_out[1];
  int K = (trans_a == false) ? dim_a[1] : dim_a[0];

  CBLAS_TRANSPOSE transA = (trans_a == false) ? CblasNoTrans : CblasTrans;
  CBLAS_TRANSPOSE transB = (trans_b == false) ? CblasNoTrans : CblasTrans;

  gemm<platform::GPUPlace, double>(
      context, transA, transB, M, N, K, alpha, matrix_a.data<double>(),
      matrix_b.data<double>(), beta, matrix_out->data<double>());
}

template <>
void batched_gemm<platform::GPUPlace, float>(
    const platform::DeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C, const int batchCount, const int strideA, const int strideB) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int strideC = M * N;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemmStridedBatched(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, strideB, A, lda, strideA,
      &beta, C, ldc, strideC, batchCount));
}

template <>
void batched_gemm<platform::GPUPlace, double>(
    const platform::DeviceContext& context, const CBLAS_TRANSPOSE transA,
    const CBLAS_TRANSPOSE transB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C, const int batchCount, const int strideA, const int strideB) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (transA == CblasNoTrans) ? K : M;
  int ldb = (transB == CblasNoTrans) ? N : K;
  int ldc = N;
  hipblasOperation_t cuTransA =
      (transA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (transB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const int strideC = M * N;

  PADDLE_ENFORCE(platform::dynload::hipblasDgemmStridedBatched(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransB, cuTransA, N, M, K, &alpha, B, ldb, strideB, A, lda, strideA,
      &beta, C, ldc, strideC, batchCount));
}

template <>
void gemv<platform::GPUPlace, float>(const platform::DeviceContext& context,
                                     const bool trans_a, const int M,
                                     const int N, const float alpha,
                                     const float* A, const float* B,
                                     const float beta, float* C) {
  hipblasOperation_t cuTransA = (trans_a == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  PADDLE_ENFORCE(platform::dynload::hipblasSgemv(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransA, N, M, &alpha, A, N, B, 1, &beta, C, 1));
}

template <>
void gemv<platform::GPUPlace, double>(const platform::DeviceContext& context,
                                      const bool trans_a, const int M,
                                      const int N, const double alpha,
                                      const double* A, const double* B,
                                      const double beta, double* C) {
  hipblasOperation_t cuTransA = (trans_a == false) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  PADDLE_ENFORCE(platform::dynload::hipblasDgemv(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      cuTransA, N, M, &alpha, A, N, B, 1, &beta, C, 1));
}

template <>
void axpy<platform::GPUPlace, float>(const platform::DeviceContext& context,
                                     const int n, const float alpha,
                                     const float* x, float* y) {
  PADDLE_ENFORCE(platform::dynload::hipblasSaxpy(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      n, &alpha, x, 1, y, 1));
}

template <>
void axpy<platform::GPUPlace, double>(const platform::DeviceContext& context,
                                      const int n, const double alpha,
                                      const double* x, double* y) {
  PADDLE_ENFORCE(platform::dynload::hipblasDaxpy(
      reinterpret_cast<const platform::CUDADeviceContext&>(context)
          .cublas_handle(),
      n, &alpha, x, 1, y, 1));
}

template struct SetConstant<platform::GPUPlace, float>;
template struct SetConstant<platform::GPUPlace, double>;
template struct SetConstant<platform::GPUPlace, int>;
template struct SetConstant<platform::GPUPlace, int64_t>;
template struct SetConstant<platform::GPUPlace, bool>;

#define DEFINE_GPU_TRANS(RANK)                                \
  template struct Transpose<platform::GPUPlace, float, RANK>; \
  template struct Transpose<platform::GPUPlace, double, RANK>;

DEFINE_GPU_TRANS(1);
DEFINE_GPU_TRANS(2);
DEFINE_GPU_TRANS(3);
DEFINE_GPU_TRANS(4);
DEFINE_GPU_TRANS(5);
DEFINE_GPU_TRANS(6);

struct TensorSetConstantGPU {
  TensorSetConstantGPU(const platform::DeviceContext& context,
                       framework::Tensor* tensor, float value)
      : context_(context), tensor_(tensor), value_(value) {}

  template <typename T>
  void operator()() const {
    SetConstant<platform::GPUPlace, T> functor;
    functor(context_, tensor_, static_cast<T>(value_));
  }

  const platform::DeviceContext& context_;
  framework::Tensor* tensor_;
  float value_;
};

template <>
void set_constant_with_place<platform::GPUPlace>(
    const platform::DeviceContext& context, framework::Tensor* tensor,
    float value) {
  framework::VisitDataType(framework::ToDataType(tensor->type()),
                           TensorSetConstantGPU(context, tensor, value));
}

template struct RowwiseAdd<platform::GPUPlace, float>;
template struct RowwiseAdd<platform::GPUPlace, double>;
template struct ColwiseSum<platform::GPUPlace, float>;
// template struct ColwiseSum<platform::GPUPlace, double>;
// The ColwiseSum<platform::GPUPlace, double> failed in debug mode,
// and only failed for this case. So reimplemented it.
template <>
void ColwiseSum<platform::GPUPlace, double>::operator()(
    const platform::DeviceContext& context, const framework::Tensor& input,
    framework::Tensor* vector) {
  auto in_dims = input.dims();
  auto size = input.numel() / in_dims[0];
  PADDLE_ENFORCE_EQ(vector->numel(), size);
  framework::Tensor one;
  one.mutable_data<double>({in_dims[0]}, context.GetPlace());
  SetConstant<platform::GPUPlace, double> set;
  set(context, &one, static_cast<double>(1.0));
  gemv<platform::GPUPlace, double>(context, true, static_cast<int>(in_dims[0]),
                                   static_cast<int>(in_dims[1]), 1.0,
                                   input.data<double>(), one.data<double>(),
                                   0.0, vector->data<double>());
}

}  // namespace math
}  // namespace operators
}  // namespace paddle
