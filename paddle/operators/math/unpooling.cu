#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/unpooling.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T>
__global__ void KernelUnpool2dMax(const int nthreads,
                                  const T* input_data,
                                  const int* indices_data,
                                  const int input_height,
                                  const int input_width,
                                  T* output_data,
                                  const int output_height,
                                  const int output_width) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = blockDim.x * gridDim.x;
  for (int i = index; i < nthreads; i += offset) {
    int out_offset =  i / (input_height * input_width) \
                      * output_height * output_width;
    int out_index = indices_data[i];
    PADDLE_ASSERT(out_index < (output_height * output_width));
    output_data[out_offset + out_index] = input_data[i];
  }
}
template <typename T>
__global__ void KernelUnpool2dMaxGrad(const int nthreads,
                                      const T* input_data,
                                      const int* indices_data,
                                      const int input_height,
                                      const int input_width,
                                      const T* output_data,
                                      const T* output_grad,
                                      const int output_height,
                                      const int output_width,
                                      T* input_grad) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    for (int i = index; i < nthreads; i += offset) {
        int out_offset =  i / (input_height * input_width) \
                          * output_height * output_width;
        int out_index = indices_data[i];
        PADDLE_ASSERT(out_index < (output_height * output_width));
        input_grad[i] = output_grad[out_offset + out_index];
    }
}
/*
 * All tensors are in NCHW format.
 */
template <typename T>
class Unpool2dMaxFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& indices,
                  framework::Tensor * output) {
    const int batch_size = input.dims()[0];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const T* input_data = input.data<T>();
    const int* indices_data = indices.data<int>();
    T* output_data = output->mutable_data<T>(context.GetPlace());

    int nthreads =  output->numel();
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelUnpool2dMax<
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(nthreads, input_data, indices_data,
                              input_height, input_width,
                              output_data, output_height, output_width);
  }
};
/*
 * All tensors are in NCHW format.
 */
template <typename T>
class Unpool2dMaxGradFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input,
                  const framework::Tensor& indices,
                  framework::Tensor * input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad) {
    const int batch_size = input.dims()[0];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const T* input_data = input.data<T>();
    const int* indices_data = indices.data<int>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad->mutable_data<T>(context.GetPlace());
    int nthreads =  output.numel();
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelUnpool2dMaxGrad<
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
                              nthreads, input_data, indices_data,
                              input_height, input_width,
                              output_data, output_grad_data,
                              output_height, output_width,
                              input_grad_data);
  }
};

template class Unpool2dMaxGradFunctor<platform::GPUPlace, float>;
template class Unpool2dMaxGradFunctor<platform::GPUPlace, double>;

template class Unpool2dMaxFunctor<platform::GPUPlace, float>;
template class Unpool2dMaxFunctor<platform::GPUPlace, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
