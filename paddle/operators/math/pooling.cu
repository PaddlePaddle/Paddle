#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/pooling.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(const int nthreads, const T* input_data,
                             T* output_data, const int channels,
                             const int input_height, const int input_width,
                             const int output_height, const int output_width,
                             const int ksize_height, const int ksize_width,
                             const int stride_height, const int stride_width,
                             const int padding_height, const int padding_width,
                             PoolProcess pool_process) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        pool_process.compute(ele, input_data[h * input_width + w]);
      }
    }
    int pool_size = (hend - hstart) * (wend - wstart);
    pool_process.finalize(ele, (static_cast<T>(pool_size)));
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, PoolProcess pool_process) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetC = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int phstart = (offsetH < ksize_height)
                      ? 0
                      : (offsetH - ksize_height) / stride_height + 1;
    int pwstart = (offsetW < ksize_width)
                      ? 0
                      : (offsetW - ksize_width) / stride_width + 1;
    int phend = min(offsetH / stride_height + 1, output_height);
    int pwend = min(offsetW / stride_width + 1, output_width);
    T gradient = 0;
    T input = input_data[index];
    int output_idx =
        (batch_idx * channels + offsetC) * output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int hstart = ph * stride_height - padding_height;
        int wstart = pw * stride_width - padding_width;
        int hend = min(hstart + ksize_height, input_height);
        int wend = min(wstart + ksize_width, input_width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = (hend - hstart) * (wend - wstart);
        int output_sub_idx = ph * output_width + pw;
        pool_process.compute(input, output_data[output_sub_idx],
                             output_grad[output_sub_idx], gradient,
                             static_cast<T>(1.0 / pool_size));
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool2DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    input_grad += (batch_idx * channels + c) * input_height * input_width;

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        if (ele == input_data[h * input_width + w]) {
          maxIndex = h * input_width + w;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      atomicAdd(input_grad + maxIndex, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
class Pool2dFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& output,
                  std::vector<int>& ksize, std::vector<int>& strides,
                  std::vector<int>& paddings, PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2D<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(nthreads, input_data, output_data, input_channels,
                              input_height, input_width, output_height,
                              output_width, ksize_height, ksize_width,
                              stride_height, stride_width, padding_height,
                              padding_width, pool_process);
  }
};

template <typename PoolProcess, typename T>
class Pool2dGradFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2DGrad<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_height, input_width, output_height, output_width,
        ksize_height, ksize_width, stride_height, stride_width, padding_height,
        padding_width, pool_process);
  }
};

template <typename T>
class MaxPool2dGradFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2DGrad<
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_height, input_width, output_height, output_width,
        ksize_height, ksize_width, stride_height, stride_width, padding_height,
        padding_width);
  }
};

template class MaxPool2dGradFunctor<platform::GPUPlace, float>;
// template class MaxPool2dGradFunctor<platform::GPUPlace, double>; // The
// 64-bit floating-point version of atomicAdd() is only supported by devices of
// compute capability 6.x and higher.

template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::MaxPoolGrad<float>, float>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::AvgPoolGrad<float>, float>;
template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::MaxPoolGrad<double>, double>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::AvgPoolGrad<double>, double>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3D(
    const int nthreads, const T* input_data, T* output_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_process) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = pool_process.initial();
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          pool_process.compute(
              ele, input_data[(d * input_height + h) * input_width + w]);
        }
      }
    }
    int pool_size = (dend - dstart) * (hend - hstart) * (wend - wstart);
    pool_process.finalize(ele, static_cast<T>(pool_size));
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_process) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetD =
        (index / input_width / input_height) % input_depth + padding_depth;
    int offsetC = (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart = (offsetD < ksize_depth)
                      ? 0
                      : (offsetD - ksize_depth) / stride_depth + 1;
    int phstart = (offsetH < ksize_height)
                      ? 0
                      : (offsetH - ksize_height) / stride_height + 1;
    int pwstart = (offsetW < ksize_width)
                      ? 0
                      : (offsetW - ksize_width) / stride_width + 1;
    int pdend = min((offsetD) / stride_depth + 1, output_depth);
    int phend = min((offsetH) / stride_height + 1, output_height);
    int pwend = min((offsetW) / stride_width + 1, output_width);

    T gradient = 0;
    T input = input_data[index];
    int output_idx = (batch_idx * channels + offsetC) * output_depth *
                     output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int dstart = pd * stride_depth - padding_depth;
          int hstart = ph * stride_height - padding_height;
          int wstart = pw * stride_width - padding_width;
          int dend = min(dstart + ksize_depth, input_depth);
          int hend = min(hstart + ksize_height, input_height);
          int wend = min(wstart + ksize_width, input_width);
          dstart = max(dstart, 0);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          int pool_size = (dend - dstart) * (hend - hstart) * (wend - wstart);
          int output_sub_idx = (pd * output_height + ph) * output_width + pw;
          pool_process.compute(input, output_data[output_sub_idx],
                               output_grad[output_sub_idx], gradient,
                               static_cast<T>(1.0 / pool_size));
        }
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool3DGrad(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height,
    const int padding_width) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    input_grad +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          if (ele == input_data[(d * input_height + h) * input_width + w]) {
            stop = true;
            maxIdx = (d * input_height + h) * input_width + w;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      atomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

template <typename PoolProcess, class T>
class Pool3dFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& output,
                  std::vector<int>& ksize, std::vector<int>& strides,
                  std::vector<int>& paddings, PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3D<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, input_channels, input_depth,
        input_height, input_width, output_depth, output_height, output_width,
        ksize_depth, ksize_height, ksize_width, stride_depth, stride_height,
        stride_width, padding_depth, padding_height, padding_width,
        pool_process);
  }
};

template <typename PoolProcess, class T>
class Pool3dGradFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings,
                  PoolProcess pool_process) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DGrad<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_depth, input_height, input_width, output_depth,
        output_height, output_width, ksize_depth, ksize_height, ksize_width,
        stride_depth, stride_height, stride_width, padding_depth,
        padding_height, padding_width, pool_process);
  }
};

template <class T>
class MaxPool3dGradFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DGrad<
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_depth, input_height, input_width, output_depth,
        output_height, output_width, ksize_depth, ksize_height, ksize_width,
        stride_depth, stride_height, stride_width, padding_depth,
        padding_height, padding_width);
  }
};

template class MaxPool3dGradFunctor<platform::GPUPlace, float>;
// template class MaxPool3dGradFunctor<platform::GPUPlace, double>;  // The
// 64-bit floating-point version of atomicAdd() is only supported by devices of
// compute capability 6.x and higher.

template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::MaxPool<float>, float>;
template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::AvgPool<float>, float>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::MaxPoolGrad<float>, float>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::AvgPoolGrad<float>, float>;
template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::MaxPool<double>, double>;
template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::AvgPool<double>, double>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::MaxPoolGrad<double>, double>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::AvgPoolGrad<double>, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
