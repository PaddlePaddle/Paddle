#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/math_function.h"
#include "paddle/operators/math/selected_rows_functor.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {
template <typename T>
struct SelectedRowsAdd<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::SelectedRows& input2,
                  framework::SelectedRows* output) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2.height());
    output->set_height(in1_height);

    auto& in1_rows = input1.rows();
    auto& in2_rows = input2.rows();
    std::vector<int64_t> out_rows;
    out_rows.reserve(in1_rows.size() + in2_rows.size());

    // concat rows
    out_rows.insert(out_rows.end(), in1_rows.begin(), in1_rows.end());
    out_rows.insert(out_rows.end(), in2_rows.begin(), in2_rows.end());
    output->set_rows(out_rows);

    auto* out_value = output->mutable_value();
    auto& in1_value = input1.value();
    auto& in2_value = input2.value();

    auto in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, in2_value.numel() / in2_rows.size());
    PADDLE_ENFORCE_EQ(in1_row_numel, out_value->numel() / out_rows.size());

    auto* out_data = out_value->data<T>();
    auto* in1_data = in1_value.data<T>();

    auto in1_place = input1.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in1_place));
    auto in2_place = input2.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in2_place));
    auto out_place = context.GetPlace();
    PADDLE_ENFORCE(platform::is_gpu_place(out_place));

    memory::Copy(
        boost::get<platform::GPUPlace>(out_place), out_data,
        boost::get<platform::GPUPlace>(in1_place), in1_data,
        in1_value.numel() * sizeof(T),
        reinterpret_cast<const platform::CUDADeviceContext&>(context).stream());

    auto* in2_data = in2_value.data<T>();
    memory::Copy(
        boost::get<platform::GPUPlace>(out_place), out_data + in1_value.numel(),
        boost::get<platform::GPUPlace>(in2_place), in2_data,
        in2_value.numel() * sizeof(T),
        reinterpret_cast<const platform::CUDADeviceContext&>(context).stream());
  }
};

template struct SelectedRowsAdd<platform::GPUPlace, float>;
template struct SelectedRowsAdd<platform::GPUPlace, double>;

namespace {
template <typename T, int block_size>
__global__ void SelectedRowsAddTensorKernel(const T* selected_rows,
                                            const int64_t* rows, T* tensor_out,
                                            int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we can not use
    // tensor_out[index] += selected_rows[index]; Instead, we have to use
    // AtomicAdd to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(tensor_out + index, selected_rows[index]);
  }
}
}  // namespace

template <typename T>
struct SelectedRowsAddTensor<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::Tensor& input2, framework::Tensor* output) {
    auto in1_height = input1.height();
    auto in2_dims = input2.dims();
    auto out_dims = output->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);
    PADDLE_ENFORCE_EQ(in1_height, out_dims[0]);

    auto& in1_value = input1.value();
    auto& in1_rows = input1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2.numel() / in1_height);
    PADDLE_ENFORCE_EQ(in1_row_numel, output->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2.data<T>();
    auto* out_data = output->data<T>();

    SetConstant<platform::GPUPlace, T> functor;
    functor(context, output, 0.0);

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(1, in1_rows.size());
    SelectedRowsAddTensorKernel<T, block_size><<<
        grid, threads, 0,
        reinterpret_cast<const platform::CUDADeviceContext&>(context)
            .stream()>>>(in1_data, in1_rows.data(), out_data, in1_row_numel);

    auto out_eigen = framework::EigenVector<T>::Flatten(*output);
    auto in2_eigen = framework::EigenVector<T>::Flatten(input2);
    out_eigen.device(*context.GetEigenDevice<platform::GPUPlace>()) =
        out_eigen + in2_eigen;
  }
};

template struct SelectedRowsAddTensor<platform::GPUPlace, float>;
template struct SelectedRowsAddTensor<platform::GPUPlace, double>;

template <typename T>
struct SelectedRowsAddTo<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& input1,
                  const int64_t input2_offset,
                  framework::SelectedRows* input2) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2->height());

    auto& in1_rows = input1.rows();
    auto& in2_rows = *(input2->mutable_rows());

    auto& in1_value = input1.value();
    auto* in2_value = input2->mutable_value();

    // concat rows
    in2_rows.insert(in2_rows.end(), in1_rows.begin(), in1_rows.end());

    auto in1_place = input1.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in1_place));
    auto in2_place = input2->place();
    PADDLE_ENFORCE(platform::is_gpu_place(in2_place));

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = in2_value->data<T>();
    memory::Copy(
        boost::get<platform::GPUPlace>(in2_place), in2_data + input2_offset,
        boost::get<platform::GPUPlace>(in1_place), in1_data,
        in1_value.numel() * sizeof(T),
        reinterpret_cast<const platform::CUDADeviceContext&>(context).stream());
  }
};

template struct SelectedRowsAddTo<platform::GPUPlace, float>;
template struct SelectedRowsAddTo<platform::GPUPlace, double>;
template struct SelectedRowsAddTo<platform::GPUPlace, int>;
template struct SelectedRowsAddTo<platform::GPUPlace, int64_t>;

namespace {
template <typename T, int block_size>
__global__ void SelectedRowsAddToTensorKernel(const T* selected_rows,
                                              const int64_t* rows,
                                              T* tensor_out,
                                              int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(tensor_out + index, selected_rows[index]);
  }
}
}  // namespace

template <typename T>
struct SelectedRowsAddToTensor<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& input1,
                  framework::Tensor* input2) {
    auto in1_height = input1.height();
    auto in2_dims = input2->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);

    auto& in1_value = input1.value();
    auto& in1_rows = input1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2->data<T>();
    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(1, in1_rows.size());
    SelectedRowsAddToTensorKernel<T, block_size><<<
        grid, threads, 0,
        reinterpret_cast<const platform::CUDADeviceContext&>(context)
            .stream()>>>(in1_data, in1_rows.data(), in2_data, in1_row_numel);
  }
};

template struct SelectedRowsAddToTensor<platform::GPUPlace, float>;
template struct SelectedRowsAddToTensor<platform::GPUPlace, double>;
template struct SelectedRowsAddToTensor<platform::GPUPlace, int>;
template struct SelectedRowsAddToTensor<platform::GPUPlace, int64_t>;
}  // namespace math
}  // namespace operators
}  // namespace paddle
