#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <set>

#include "paddle/operators/math/math_function.h"
#include "paddle/operators/math/selected_rows_functor.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {
template <typename T>
struct SelectedRowsAdd<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::SelectedRows& input2,
                  framework::SelectedRows* output) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2.height());
    output->set_height(in1_height);

    framework::Vector<int64_t> in1_rows(input1.rows());
    auto& in2_rows = input2.rows();
    std::vector<int64_t> out_rows;
    out_rows.reserve(in1_rows.size() + in2_rows.size());

    // concat rows
    out_rows.insert(out_rows.end(), in1_rows.begin(), in1_rows.end());
    out_rows.insert(out_rows.end(), in2_rows.begin(), in2_rows.end());
    output->set_rows(out_rows);

    auto* out_value = output->mutable_value();
    auto& in1_value = input1.value();
    auto& in2_value = input2.value();

    auto in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, in2_value.numel() / in2_rows.size());
    PADDLE_ENFORCE_EQ(in1_row_numel, out_value->numel() / out_rows.size());

    auto* out_data = out_value->data<T>();
    auto* in1_data = in1_value.data<T>();

    auto in1_place = input1.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in1_place));
    auto in2_place = input2.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in2_place));
    auto out_place = context.GetPlace();
    PADDLE_ENFORCE(platform::is_gpu_place(out_place));

    memory::Copy(
        boost::get<platform::CUDAPlace>(out_place), out_data,
        boost::get<platform::CUDAPlace>(in1_place), in1_data,
        in1_value.numel() * sizeof(T),
        reinterpret_cast<const platform::CUDADeviceContext&>(context).stream());

    auto* in2_data = in2_value.data<T>();
    memory::Copy(boost::get<platform::CUDAPlace>(out_place),
                 out_data + in1_value.numel(),
                 boost::get<platform::CUDAPlace>(in2_place), in2_data,
                 in2_value.numel() * sizeof(T), context.stream());
  }
};

template struct SelectedRowsAdd<platform::CUDADeviceContext, float>;
template struct SelectedRowsAdd<platform::CUDADeviceContext, double>;

namespace {
template <typename T, int block_size>
__global__ void SelectedRowsAddTensorKernel(const T* selected_rows,
                                            const int64_t* rows, T* tensor_out,
                                            int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we can not use
    // tensor_out[index] += selected_rows[index]; Instead, we have to use
    // AtomicAdd to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(tensor_out + index, selected_rows[index]);
  }
}
}  // namespace

template <typename T>
struct SelectedRowsAddTensor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  const framework::Tensor& input2, framework::Tensor* output) {
    auto in1_height = input1.height();
    auto in2_dims = input2.dims();
    auto out_dims = output->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);
    PADDLE_ENFORCE_EQ(in1_height, out_dims[0]);

    auto& in1_value = input1.value();
    framework::Vector<int64_t> in1_rows(input1.rows());

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2.numel() / in1_height);
    PADDLE_ENFORCE_EQ(in1_row_numel, output->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2.data<T>();
    auto* out_data = output->data<T>();

    SetConstant<platform::CUDADeviceContext, T> functor;
    functor(context, output, 0.0);

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(1, in1_rows.size());
    SelectedRowsAddTensorKernel<
        T, block_size><<<grid, threads, 0, context.stream()>>>(
        in1_data, in1_rows.cuda_data(), out_data, in1_row_numel);

    auto out_eigen = framework::EigenVector<T>::Flatten(*output);
    auto in2_eigen = framework::EigenVector<T>::Flatten(input2);
    out_eigen.device(*context.eigen_device()) = out_eigen + in2_eigen;
  }
};

template struct SelectedRowsAddTensor<platform::CUDADeviceContext, float>;
template struct SelectedRowsAddTensor<platform::CUDADeviceContext, double>;

template <typename T>
struct SelectedRowsAddTo<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  const int64_t input2_offset,
                  framework::SelectedRows* input2) {
    auto in1_height = input1.height();
    PADDLE_ENFORCE_EQ(in1_height, input2->height());

    framework::Vector<int64_t> in1_rows(input1.rows());
    auto& in2_rows = *(input2->mutable_rows());

    auto& in1_value = input1.value();
    auto* in2_value = input2->mutable_value();

    // concat rows
    in2_rows.insert(in2_rows.end(), in1_rows.begin(), in1_rows.end());

    auto in1_place = input1.place();
    PADDLE_ENFORCE(platform::is_gpu_place(in1_place));
    auto in2_place = input2->place();
    PADDLE_ENFORCE(platform::is_gpu_place(in2_place));

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = in2_value->data<T>();
    memory::Copy(boost::get<platform::CUDAPlace>(in2_place),
                 in2_data + input2_offset,
                 boost::get<platform::CUDAPlace>(in1_place), in1_data,
                 in1_value.numel() * sizeof(T), context.stream());
  }
};

template struct SelectedRowsAddTo<platform::CUDADeviceContext, float>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext, double>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext, int>;
template struct SelectedRowsAddTo<platform::CUDADeviceContext, int64_t>;

namespace {
template <typename T, int block_size>
__global__ void SelectedRowsAddToTensorKernel(const T* selected_rows,
                                              const int64_t* rows,
                                              T* tensor_out,
                                              int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(tensor_out + index, selected_rows[index]);
  }
}
}  // namespace

template <typename T>
struct SelectedRowsAddToTensor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::SelectedRows& input1,
                  framework::Tensor* input2) {
    auto in1_height = input1.height();
    auto in2_dims = input2->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);

    auto& in1_value = input1.value();
    framework::Vector<int64_t> in1_rows(input1.rows());

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2->numel() / in1_height);

    auto* in1_data = in1_value.data<T>();
    auto* in2_data = input2->data<T>();
    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid(1, in1_rows.size());
    SelectedRowsAddToTensorKernel<
        T, block_size><<<grid, threads, 0, context.stream()>>>(
        in1_data, in1_rows.cuda_data(), in2_data, in1_row_numel);
  }
};

template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, float>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, double>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, int>;
template struct SelectedRowsAddToTensor<platform::CUDADeviceContext, int64_t>;

namespace scatter {

template <typename T, int block_size>
__global__ void MergeAddKernel(const T* input, const int64_t* input_rows,
                               T* out, const int64_t* out_rows,
                               size_t out_rows_size, int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;
  __shared__ size_t out_idx;

  if (tid == 0) {
    for (size_t i = 0; i < out_rows_size; i++) {
      if (input_rows[ty] == out_rows[i]) {
        out_idx = i;
      }
    }
  }

  __syncthreads();

  input += ty * row_numel;
  out += out_idx * row_numel;
  for (int index = tid; index < row_numel; index += block_size) {
    paddle::platform::CudaAtomicAdd(out + index, input[index]);
  }
}

template <typename T>
struct MergeAdd<platform::CUDADeviceContext, T> {
  framework::SelectedRows operator()(const platform::CUDADeviceContext& context,
                                     const framework::SelectedRows& input) {
    framework::SelectedRows out;
    framework::Vector<int64_t> input_rows(input.rows());
    std::set<int64_t> row_set(input_rows.begin(), input_rows.end());
    std::vector<int64_t> merge_rows(row_set.begin(), row_set.end());

    auto input_width = input.value().dims()[1];

    out.set_rows(merge_rows);
    out.set_height(input.height());
    out.mutable_value()->mutable_data<T>(
        framework::make_ddim(
            {static_cast<int64_t>(merge_rows.size()), input_width}),
        context.GetPlace());

    math::SetConstant<platform::CUDADeviceContext, T> constant_functor;
    constant_functor(context, out.mutable_value(), 0.0);

    auto* out_data = out.mutable_value()->data<T>();
    auto* input_data = input.value().data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid1(1, input_rows.size());

    MergeAddKernel<
        T, 256><<<grid1, threads, 0,
                  reinterpret_cast<const platform::CUDADeviceContext&>(context)
                      .stream()>>>(input_data, input_rows.cuda_data(), out_data,
                                   out.mutable_rows()->cuda_data(),
                                   out.rows().size(), input_width);
    return out;
  }
};

template struct MergeAdd<platform::CUDADeviceContext, float>;
template struct MergeAdd<platform::CUDADeviceContext, double>;
template struct MergeAdd<platform::CUDADeviceContext, int>;
template struct MergeAdd<platform::CUDADeviceContext, int64_t>;

template <typename T, int block_size>
__global__ void UpdateToTensorKernel(const T* selected_rows,
                                     const int64_t* rows, const ScatterOps& op,
                                     T* tensor_out, int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;
  // FIXME(typhoonzero): use macro fix the below messy code.
  switch (op) {
    case ScatterOps::ASSIGN:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] = selected_rows[index];
      }
      break;
    case ScatterOps::ADD:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] += selected_rows[index];
      }
      break;
    case ScatterOps::SUB:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] -= selected_rows[index];
      }
      break;
    case ScatterOps::SUBBY:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] = selected_rows[index] - tensor_out[index];
      }
      break;
    case ScatterOps::MUL:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] *= selected_rows[index];
      }
      break;
    case ScatterOps::DIV:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] /= selected_rows[index];
      }
      break;
    case ScatterOps::DIVBY:
      for (int index = tid; index < row_numel; index += block_size) {
        tensor_out[index] = selected_rows[index] / tensor_out[index];
      }
      break;
  }
}

template <typename T>
struct UpdateToTensor<platform::CUDADeviceContext, T> {
  void operator()(const platform::CUDADeviceContext& context,
                  const ScatterOps& op, const framework::SelectedRows& input1,
                  framework::Tensor* input2) {
    // NOTE: Use SelectedRowsAddToTensor for better performance
    //       no additional MergeAdd called.
    MergeAdd<platform::CUDADeviceContext, T> merge_func;
    auto merged_in1 = merge_func(context, input1);

    auto in1_height = merged_in1.height();
    auto in2_dims = input2->dims();
    PADDLE_ENFORCE_EQ(in1_height, in2_dims[0]);

    auto& in1_value = merged_in1.value();
    auto& in1_rows = merged_in1.rows();

    int64_t in1_row_numel = in1_value.numel() / in1_rows.size();
    PADDLE_ENFORCE_EQ(in1_row_numel, input2->numel() / in1_height);

    auto* in1_data = in1_value.template data<T>();
    auto* in2_data = input2->data<T>();

    dim3 threads(platform::PADDLE_CUDA_NUM_THREADS, 1);
    dim3 grid(1, in1_rows.size());
    UpdateToTensorKernel<T, platform::PADDLE_CUDA_NUM_THREADS><<<
        grid, threads, 0, context.stream()>>>(in1_data, in1_rows.cuda_data(),
                                              op, in2_data, in1_row_numel);
  }
};
}  // namespace scatter
}  // namespace math
}  // namespace operators
}  // namespace paddle
