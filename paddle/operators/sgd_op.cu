#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/sgd_op.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

namespace {

template <typename T>
__global__ void SGDKernel(const T* g, const T* p, const T* learning_rate,
                          const int num, T* p_out) {
  T lr = learning_rate[0];
  int grid_size = blockDim.x * gridDim.x;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += grid_size) {
    T g_data = g[i];
    T p_data = p[i];
    p_out[i] = p_data - lr * g_data;
  }
}

template <typename T, int block_size>
__global__ void SparseSGDFunctorKernel(const T* selected_rows,
                                       const int64_t* rows,
                                       const T* learning_rate, T* tensor_out,
                                       int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  selected_rows += ty * row_numel;
  tensor_out += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(
        tensor_out + index, -1.0 * learning_rate[0] * selected_rows[index]);
  }
}
}  // namespace

template <typename T>
class SGDOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* param = ctx.Input<framework::Tensor>("Param");
    auto* param_out = ctx.Output<framework::Tensor>("ParamOut");
    auto* learning_rate = ctx.Input<framework::Tensor>("LearningRate");

    auto* grad_var = ctx.InputVar("Grad");
    // Actually, all tensors are LoDTensor except SelectedRows.
    if (grad_var->IsType<framework::LoDTensor>()) {
      param_out->mutable_data<T>(ctx.GetPlace());
      auto* grad = ctx.Input<framework::Tensor>("Grad");
      auto* grad_data = grad->data<T>();
      auto* param_data = param->data<T>();
      auto* param_out_data = param_out->data<T>();

      int block = 512;
      int grid = (param->numel() + block - 1) / block;

      SGDKernel<T><<<grid, block, 0, ctx.cuda_device_context().stream()>>>(
          grad_data, param_data, learning_rate->data<T>(), param->numel(),
          param_out_data);

    } else if (grad_var->IsType<framework::SelectedRows>()) {
      // TODO(qijun): In Sparse SGD operator, in-place update is enforced.
      // This manual optimization brings difficulty to track data dependency.
      // It's better to find a more elegant solution.
      PADDLE_ENFORCE_EQ(param, param_out);
      auto* grad = ctx.Input<framework::SelectedRows>("Grad");

      auto in_height = grad->height();
      auto out_dims = param_out->dims();
      PADDLE_ENFORCE_EQ(in_height, out_dims[0]);

      auto& in_value = grad->value();
      framework::Vector<int64_t> in_rows(grad->rows());

      int64_t in_row_numel = in_value.numel() / in_rows.size();
      PADDLE_ENFORCE_EQ(in_row_numel, param_out->numel() / in_height);

      auto* in_data = in_value.data<T>();
      auto* out_data = param_out->data<T>();

      const int block_size = 256;
      dim3 threads(block_size, 1);
      dim3 grid(1, in_rows.size());
      SparseSGDFunctorKernel<
          T, 256><<<grid, threads, 0, ctx.cuda_device_context().stream()>>>(
          in_data, in_rows.cuda_data(), learning_rate->data<T>(), out_data,
          in_row_numel);

    } else {
      PADDLE_THROW("Unsupported Variable Type of Grad");
    }
  }
};
}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(sgd, ops::SGDOpCUDAKernel<float>,
                        ops::SGDOpCUDAKernel<double>);
