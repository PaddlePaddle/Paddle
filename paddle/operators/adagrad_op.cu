#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/operators/adagrad_op.h"
#include "paddle/operators/math/math_function.h"
#include "paddle/operators/math/selected_rows_functor.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

namespace {

template <typename T, int block_size>
__global__ void MergeGradKernel(const T* grad, const int64_t* grad_rows,
                                T* grad_merge, const int64_t* grad_merge_rows,
                                size_t grad_merge_rows_size,
                                int64_t row_numel) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;
  __shared__ size_t grad_merge_idx;

  if (tid == 0) {
    for (size_t i = 0; i < grad_merge_rows_size; i++) {
      if (grad_rows[ty] == grad_merge_rows[i]) {
        grad_merge_idx = i;
      }
    }
  }

  __syncthreads();

  grad += ty * row_numel;
  grad_merge += grad_merge_idx * row_numel;
  for (int index = tid; index < row_numel; index += block_size) {
    paddle::platform::CudaAtomicAdd(grad_merge + index, grad[index]);
  }
}

template <typename T, int block_size>
__global__ void SparseAdagradFunctorKernel(const T* grad, const int64_t* rows,
                                           const T* learning_rate, T* param,
                                           T* moment, int64_t row_numel,
                                           T epsilon) {
  const int ty = blockIdx.y;
  int tid = threadIdx.x;

  grad += ty * row_numel;
  param += rows[ty] * row_numel;
  moment += rows[ty] * row_numel;

  for (int index = tid; index < row_numel; index += block_size) {
    // Since index in rows of SelectedRows can be duplicate, we have to use
    // Atomic Operation to avoid concurrent write error.
    paddle::platform::CudaAtomicAdd(param + index,
                                    -1.0 * learning_rate[0] * grad[index] /
                                        (sqrt(moment[index]) + epsilon));
  }
}
}  // namespace

template <typename T>
struct SparseAdagradFunctor<platform::GPUPlace, T> {
  void operator()(const platform::DeviceContext& context,
                  const framework::SelectedRows& grad,
                  const framework::Tensor& learning_rate, T epsilon,
                  framework::Tensor* moment, framework::Tensor* param) {
    // 1. g_m.rows = set(g.rows)
    auto grad_rows = grad.rows();
    std::set<int64_t> row_set(grad_rows.begin(), grad_rows.end());
    std::vector<int64_t> merge_rows(row_set.begin(), row_set.end());

    auto grad_width = grad.value().dims()[1];
    std::unique_ptr<framework::SelectedRows> grad_merge{
        new framework::SelectedRows()};
    grad_merge->set_rows(merge_rows);
    grad_merge->set_height(grad.height());
    grad_merge->mutable_value()->mutable_data<T>(
        framework::make_ddim(
            {static_cast<int64_t>(merge_rows.size()), grad_width}),
        context.GetPlace());

    math::SetConstant<platform::GPUPlace, T> constant_functor;
    constant_functor(context, grad_merge->mutable_value(), 0.0);

    auto* grad_merge_data = grad_merge->mutable_value()->data<T>();
    auto* grad_data = grad.value().data<T>();

    const int block_size = 256;
    dim3 threads(block_size, 1);
    dim3 grid1(1, grad_rows.size());

    MergeGradKernel<
        T, 256><<<grid1, threads, 0,
                  reinterpret_cast<const platform::CUDADeviceContext&>(context)
                      .stream()>>>(grad_data, grad.rows().data(),
                                   grad_merge_data, grad_merge->rows().data(),
                                   grad_merge->rows().size(), grad_width);

    // 2. m += g_m * g_m
    std::unique_ptr<framework::SelectedRows> grad_square{
        new framework::SelectedRows()};
    grad_square->set_rows(grad_merge->rows());
    grad_square->set_height(grad_merge->height());
    grad_square->mutable_value()->mutable_data<T>(grad_merge->value().dims(),
                                                  context.GetPlace());
    auto gs =
        framework::EigenVector<T>::Flatten(*(grad_square->mutable_value()));
    auto gm = framework::EigenVector<T>::Flatten(grad_merge->value());
    gs.device(*context.GetEigenDevice<platform::GPUPlace>()) = gm * gm;

    math::SelectedRowsAddToTensor<platform::GPUPlace, T> functor;
    functor(context, *grad_square, moment);

    // 3. update parameter
    auto* lr = learning_rate.data<T>();
    auto* param_data = param->data<T>();
    auto* moment_data = moment->data<T>();

    dim3 grid2(1, merge_rows.size());
    SparseAdagradFunctorKernel<
        T, 256><<<grid2, threads, 0,
                  reinterpret_cast<const platform::CUDADeviceContext&>(context)
                      .stream()>>>(grad_merge_data, grad_merge->rows().data(),
                                   lr, param_data, moment_data, grad_width,
                                   epsilon);
  }
};

template struct SparseAdagradFunctor<platform::GPUPlace, float>;
template struct SparseAdagradFunctor<platform::GPUPlace, double>;

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(
    adagrad, ops::AdagradOpKernel<paddle::platform::GPUPlace, float>,
    ops::AdagradOpKernel<paddle::platform::GPUPlace, double>);
