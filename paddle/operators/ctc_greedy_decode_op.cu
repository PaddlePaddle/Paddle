#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "paddle/operators/ctc_greedy_decode_op.h"
#include "paddle/platform/cuda_helper.h"
#include "paddle/platform/gpu_info.h"

namespace paddle {
namespace operators {
using platform::PADDLE_CUDA_NUM_THREADS;

__device__ static float atomicMaxF(float* address, float val) {
  int* address_as_i = (int*)address;
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = ::atomicCAS(address_as_i, assumed,
                      __float_as_int(::fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

template <typename T, int BlockSize>
__global__ void ArgmaxCudaKernel(const size_t seq_width, const T* logits,
                                 int* output) {
  T local_max_value = 0;
  int local_max_index = 0;
  __shared__ T max_value;
  if (threadIdx.x == 0) {
    max_value = 0;
  }
  __syncthreads();

  for (int i = threadIdx.x; i < seq_width; i += BlockSize) {
    T value = logits[blockIdx.x * seq_width + i];
    if (value > local_max_value) {
      local_max_value = value;
      local_max_index = i;
    }
  }

  atomicMaxF(&max_value, local_max_value);

  __syncthreads();

  if (local_max_value == max_value) {
    output[blockIdx.x] = local_max_index;
  }
}

template <typename T>
__global__ void MergeAndDelCudaKernel(const int64_t num_token, int* tokens,
                                      const size_t num_seq, size_t* lod0,
                                      const int blank, const int merge_repeated,
                                      size_t* out_lod0, int* output) {
  int ouput_idx = 0;
  out_lod0[0] = 0;

  for (int i = 0; i < num_seq; ++i) {
    int pre_token = -1;
    for (int j = lod0[i]; j < lod0[i + 1]; ++j) {
      if (tokens[j] != blank && !(merge_repeated && tokens[j] == pre_token)) {
        output[ouput_idx] = tokens[j];
        ++ouput_idx;
      }
      pre_token = tokens[j];
    }
    out_lod0[i + 1] = ouput_idx;
  }
}

template <typename T>
class CTCGreedyDecodeOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use CUDAPlace.");
    auto* input = ctx.Input<LoDTensor>("Input");
    auto* output = ctx.Output<LoDTensor>("Output");

    const int64_t num_tokens = input->dims()[0];
    const size_t seq_width = input->numel() / num_tokens;
    const T* logits = input->data<T>();
    Tensor tmp;
    int* tokens = tmp.mutable_data<int>({num_tokens, 1}, ctx.GetPlace());
    // get argmax
    // platform::GpuMemsetAsync(args, 0, sizeof(float), stream);

    auto stream = ctx.cuda_device_context().stream();
    ArgmaxCudaKernel<T, PADDLE_CUDA_NUM_THREADS><<<
        num_tokens, PADDLE_CUDA_NUM_THREADS, 0, stream>>>(seq_width, logits,
                                                          tokens);

    const size_t level = 0;
    auto input_lod = framework::ToAbsOffset(input->lod());
    const size_t num_seq = input_lod[level].size() - 1;
    const int blank = ctx.Attr<int>("blank");
    const int merge_repeated =
        static_cast<int>(ctx.Attr<bool>("merge_repeated"));

    thrust::device_vector<size_t> dev_out_lod0(input_lod[level].size());
    size_t* dev_out_lod0_ptr = thrust::raw_pointer_cast(dev_out_lod0.data());

    int* output_data =
        output->mutable_data<int>({num_tokens, 1}, ctx.GetPlace());
    MergeAndDelCudaKernel<T><<<1, 1, 0, stream>>>(
        num_tokens, tokens, num_seq, input_lod[level].data(), blank,
        merge_repeated, dev_out_lod0_ptr, output_data);

    thrust::host_vector<size_t> host_out_lod0(dev_out_lod0.begin(),
                                              dev_out_lod0.end());
    framework::LoD out_lod;
    out_lod.push_back(host_out_lod0);
    output->set_lod(out_lod);

    output->Resize({static_cast<int64_t>(host_out_lod0.back()), 1});
  }
};

}  // namespace operators
}  // namespace paddle

REGISTER_OP_CUDA_KERNEL(ctc_greedy_decode,
                        paddle::operators::CTCGreedyDecodeOpCUDAKernel<float>);
