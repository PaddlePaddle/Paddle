#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/pool_cudnn_op.h"
#include "paddle/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedPoolingDescriptor = platform::ScopedPoolingDescriptor;
using DataLayout = platform::DataLayout;
using PoolingMode = platform::PoolingMode;

template <typename T>
class PoolCudnnOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    const Tensor *input = ctx.Input<Tensor>("X");
    Tensor *output = ctx.Output<Tensor>("Out");

    const T *input_data = input->data<T>();
    T *output_data = output->mutable_data<T>(ctx.GetPlace());

    std::string pooling_type = ctx.Attr<std::string>("pooling_type");
    std::vector<int> ksize = ctx.Attr<std::vector<int>>("ksize");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    if (ctx.Attr<bool>("global_pooling")) {
      for (size_t i = 0; i < ksize.size(); ++i) {
        paddings[i] = 0;
        ksize[i] = static_cast<int>(input->dims()[i + 2]);
      }
    }

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedPoolingDescriptor pool_desc;
    DataLayout layout = DataLayout::kNCHW;

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize2int(input->dims()));
    hipdnnTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
        layout, framework::vectorize2int(output->dims()));

    PoolingMode pooling_mode;
    if (pooling_type == "max") {
      pooling_mode = PoolingMode::kMaximum;
    } else {
      pooling_mode = PoolingMode::kAverage;
    }

    hipdnnPoolingDescriptor_t cudnn_pool_desc =
        pool_desc.descriptor(pooling_mode, ksize, paddings, strides);

    // ------------------- cudnn pool algorithm ---------------------
    auto handle = ctx.cuda_device_context().cudnn_handle();
    T alpha = 1.0f, beta = 0.0f;

    PADDLE_ENFORCE(platform::dynload::hipdnnPoolingForward(
        handle, cudnn_pool_desc, &alpha, cudnn_input_desc, input_data, &beta,
        cudnn_output_desc, output_data));
  }
};

template <typename T>
class PoolCudnnGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    const Tensor *input = ctx.Input<Tensor>("X");
    const Tensor *output = ctx.Input<Tensor>("Out");
    const Tensor *output_grad =
        ctx.Input<Tensor>(framework::GradVarName("Out"));
    Tensor *input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    std::string pooling_type = ctx.Attr<std::string>("pooling_type");
    std::vector<int> ksize = ctx.Attr<std::vector<int>>("ksize");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");

    if (ctx.Attr<bool>("global_pooling")) {
      for (size_t i = 0; i < ksize.size(); ++i) {
        paddings[i] = 0;
        ksize[i] = static_cast<int>(input->dims()[i + 2]);
      }
    }

    const T *input_data = input->data<T>();
    const T *output_data = output->data<T>();
    const T *output_grad_data = output_grad->data<T>();

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedPoolingDescriptor pool_desc;
    DataLayout layout = DataLayout::kNCHW;

    hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
        layout, framework::vectorize2int(input->dims()));
    hipdnnTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
        layout, framework::vectorize2int(output->dims()));

    PoolingMode pooling_mode;
    if (pooling_type == "max") {
      pooling_mode = PoolingMode::kMaximum;
    } else {
      pooling_mode = PoolingMode::kAverage;
    }

    hipdnnPoolingDescriptor_t cudnn_pool_desc =
        pool_desc.descriptor(pooling_mode, ksize, paddings, strides);

    // ------------------- cudnn pool algorithm ---------------------
    auto handle = ctx.cuda_device_context().cudnn_handle();
    T alpha = 1.0f, beta = 0.0f;

    if (input_grad) {
      T *input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());
      math::SetConstant<paddle::platform::GPUPlace, T> set_zero;
      set_zero(ctx.device_context(), input_grad, static_cast<T>(0));

      PADDLE_ENFORCE(platform::dynload::hipdnnPoolingBackward(
          handle, cudnn_pool_desc, &alpha, cudnn_output_desc, output_data,
          cudnn_output_desc, output_grad_data, cudnn_input_desc, input_data,
          &beta, cudnn_input_desc, input_grad_data));
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_GPU_KERNEL(pool2d_cudnn, ops::PoolCudnnOpKernel<float>);
REGISTER_OP_GPU_KERNEL(pool2d_cudnn_grad, ops::PoolCudnnGradOpKernel<float>);
