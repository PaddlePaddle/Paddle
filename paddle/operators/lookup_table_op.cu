#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/framework/eigen.h"
#include "paddle/framework/op_registry.h"
#include "paddle/operators/lookup_table_op.h"
#include "paddle/platform/assert.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

template <typename T, int BlockDimX, int BlockDimY, int GridDimX,
          bool PaddingFlag>
__global__ void LookupTable(T* output, const T* table, const int64_t* ids,
                            const int64_t N, const int64_t K, const int64_t D,
                            const int64_t padding_idx) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int64_t id = ids[idy];
    PADDLE_ASSERT(id >= 0);
    PADDLE_ASSERT(id < N);
    T* out = output + idy * D;
    const T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      if (PaddingFlag) {
        if (id == padding_idx)
          out[i] = static_cast<T>(0);
        else
          out[i] = tab[i];
      } else {
        out[i] = tab[i];
      }
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void LookupTableGrad(T* table, const T* output, const int64_t* ids,
                                const int64_t N, const int64_t K,
                                const int64_t D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int id = ids[idy];
    PADDLE_ASSERT(id >= 0);
    PADDLE_ASSERT(id < N);
    const T* out = output + idy * D;
    T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      paddle::platform::CudaAtomicAdd(&tab[i], out[i]);
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T>
class LookupTableCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* table_t = context.Input<LoDTensor>("W");
    auto* ids_t = context.Input<LoDTensor>("Ids");
    auto* output_t = context.Output<LoDTensor>("Out");
    int64_t padding_idx = context.Attr<int64_t>("padding_idx");

    size_t N = table_t->dims()[0];
    size_t D = table_t->dims()[1];
    size_t K = ids_t->numel();
    auto* ids = ids_t->data<int64_t>();
    auto* table = table_t->data<T>();
    auto* output = output_t->mutable_data<T>(context.GetPlace());

    dim3 threads(128, 8);
    dim3 grids(8, 1);

    if (padding_idx == -1)
      LookupTable<
          T, 128, 8, 8,
          false><<<grids, threads, 0, context.cuda_device_context().stream()>>>(
          output, table, ids, N, K, D, padding_idx);
    else
      LookupTable<
          T, 128, 8, 8,
          true><<<grids, threads, 0, context.cuda_device_context().stream()>>>(
          output, table, ids, N, K, D, padding_idx);
  }
};

template <typename T>
class LookupTableGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto& dev_ctx =
        context.template device_context<platform::CUDADeviceContext>();
    bool is_sparse = context.Attr<bool>("is_sparse");
    // Since paddings are not trainable and fixed in forward, the gradient of
    // paddings makes no sense and we don't deal with it in backward.
    if (is_sparse) {
      auto* ids = context.Input<LoDTensor>("Ids");
      auto* table = context.Input<LoDTensor>("W");
      auto* d_output = context.Input<LoDTensor>(framework::GradVarName("Out"));
      auto* d_table = context.Output<SelectedRows>(framework::GradVarName("W"));

      auto* ids_data = ids->data<int64_t>();
      auto ids_dim = ids->dims();

      auto stream = dev_ctx.stream();
      // copy GPU memory to CPU pinned memory
      framework::Vector<int64_t> new_rows;
      new_rows.resize(ids_dim[0]);
      auto gpu_place = boost::get<platform::CUDAPlace>(context.GetPlace());

      memory::Copy(platform::CPUPlace(), new_rows.cuda_data(), gpu_place,
                   ids_data, ids_dim[0] * sizeof(int64_t), stream);

      d_table->set_rows(new_rows);

      auto* d_table_value = d_table->mutable_value();
      d_table_value->Resize({ids_dim[0], table->dims()[1]});
      d_table_value->mutable_data<T>(context.GetPlace());

      auto* d_table_data = d_table_value->data<T>();
      auto* d_output_data = d_output->data<T>();
      PADDLE_ENFORCE_EQ(d_table_value->dims(), d_output->dims());
      memory::Copy(gpu_place, d_table_data, gpu_place, d_output_data,
                   d_output->numel() * sizeof(T), stream);

    } else {
      auto ids_t = context.Input<LoDTensor>("Ids");
      auto d_output_t = context.Input<LoDTensor>(framework::GradVarName("Out"));
      auto d_table_t = context.Output<LoDTensor>(framework::GradVarName("W"));

      int N = d_table_t->dims()[0];
      int D = d_table_t->dims()[1];
      int K = ids_t->numel();
      const int64_t* ids = ids_t->data<int64_t>();
      const T* d_output = d_output_t->data<T>();
      T* d_table = d_table_t->mutable_data<T>(context.GetPlace());

      auto t = framework::EigenVector<T>::Flatten(*d_table_t);
      t.device(*dev_ctx.eigen_device()) = t.constant(static_cast<T>(0));

      dim3 threads(128, 8);
      dim3 grids(8, 1);
      LookupTableGrad<T, 128, 8, 8><<<grids, threads, 0, dev_ctx.stream()>>>(
          d_table, d_output, ids, N, K, D);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(lookup_table, ops::LookupTableCUDAKernel<float>,
                        ops::LookupTableCUDAKernel<double>);
REGISTER_OP_CUDA_KERNEL(lookup_table_grad,
                        ops::LookupTableGradCUDAKernel<float>,
                        ops::LookupTableGradCUDAKernel<double>);
