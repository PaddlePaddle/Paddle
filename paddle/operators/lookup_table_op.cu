#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License. */

#include "paddle/framework/eigen.h"
#include "paddle/framework/op_registry.h"
#include "paddle/platform/assert.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void LookupTable(T* output, const T* table, const int32_t* ids,
                            const int N, const int K, const int D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int id = ids[idy];
    PADDLE_ASSERT(id >= 0);
    PADDLE_ASSERT(id < N);
    T* out = output + idy * D;
    const T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      out[i] = tab[i];
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void LookupTableGrad(T* table, const T* output, const int32_t* ids,
                                const int N, const int K, const int D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int id = ids[idy];
    PADDLE_ASSERT(id >= 0);
    PADDLE_ASSERT(id < N);
    const T* out = output + idy * D;
    T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      paddle::platform::CudaAtomicAdd(&tab[i], out[i]);
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T>
class LookupTableCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto table_t = context.Input<Tensor>("W");
    auto ids_t = context.Input<Tensor>("Ids");
    auto output_t = context.Output<Tensor>("Out");

    size_t N = table_t->dims()[0];
    size_t D = table_t->dims()[1];
    size_t K = ids_t->numel();
    auto ids = ids_t->data<int32_t>();
    auto table = table_t->data<T>();
    auto output = output_t->mutable_data<T>(context.GetPlace());

    dim3 threads(128, 8);
    dim3 grids(8, 1);
    LookupTable<T, 128, 8, 8><<<
        grids, threads, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                               context.device_context())
                               .stream()>>>(output, table, ids, N, K, D);
  }
};

template <typename T>
class LookupTableGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto ids_t = context.Input<Tensor>("Ids");
    auto d_output_t = context.Input<Tensor>(framework::GradVarName("Out"));
    auto d_table_t = context.Output<Tensor>(framework::GradVarName("W"));

    int N = d_table_t->dims()[0];
    int D = d_table_t->dims()[1];
    int K = ids_t->numel();
    const int32_t* ids = ids_t->data<int32_t>();
    const T* d_output = d_output_t->data<T>();
    T* d_table = d_table_t->mutable_data<T>(context.GetPlace());

    auto t = framework::EigenVector<T>::Flatten(*d_table_t);
    t.device(context.GetEigenDevice<platform::GPUPlace>()) =
        t.constant(static_cast<T>(0));

    dim3 threads(128, 8);
    dim3 grids(8, 1);
    LookupTableGrad<T, 128, 8, 8><<<
        grids, threads, 0, reinterpret_cast<const platform::CUDADeviceContext&>(
                               context.device_context())
                               .stream()>>>(d_table, d_output, ids, N, K, D);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_GPU_KERNEL(lookup_table, ops::LookupTableCUDAKernel<float>);
REGISTER_OP_GPU_KERNEL(lookup_table_grad,
                       ops::LookupTableGradCUDAKernel<float>);
